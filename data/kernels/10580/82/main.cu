#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "__ADAGradn.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
int nrows = 1;
int ncols = 1;
float *mm = NULL;
hipMalloc(&mm, XSIZE*YSIZE);
float *um = NULL;
hipMalloc(&um, XSIZE*YSIZE);
float *ssq = NULL;
hipMalloc(&ssq, XSIZE*YSIZE);
float *momentum = NULL;
hipMalloc(&momentum, XSIZE*YSIZE);
float mu = 1;
float *mask = NULL;
hipMalloc(&mask, XSIZE*YSIZE);
int maskr = 1;
float nw = 1;
float *ve = NULL;
hipMalloc(&ve, XSIZE*YSIZE);
int nve = 1;
float *ts = NULL;
hipMalloc(&ts, XSIZE*YSIZE);
int nts = 1;
float *lr = NULL;
hipMalloc(&lr, XSIZE*YSIZE);
int nlr = 1;
float langevin = 1;
float eps = 1;
int doupdate = 1;
hiprandState *rstates = NULL;
hipMalloc(&rstates, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
__ADAGradn<<<gridBlock,threadBlock>>>(nrows,ncols,mm,um,ssq,momentum,mu,mask,maskr,nw,ve,nve,ts,nts,lr,nlr,langevin,eps,doupdate,rstates);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
__ADAGradn<<<gridBlock,threadBlock>>>(nrows,ncols,mm,um,ssq,momentum,mu,mask,maskr,nw,ve,nve,ts,nts,lr,nlr,langevin,eps,doupdate,rstates);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
__ADAGradn<<<gridBlock,threadBlock>>>(nrows,ncols,mm,um,ssq,momentum,mu,mask,maskr,nw,ve,nve,ts,nts,lr,nlr,langevin,eps,doupdate,rstates);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}