#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Compute_Path(int *Md, const int Width, const int k)
{
    // Calculate global thread coordinates
    int ROW = blockIdx.x * blockDim.x + threadIdx.x;

    // Check within bounds
    if (ROW < Width) {
        // Pre-compute reused index values
        int index_row_k = ROW * Width + k;
        int index_k_col = k * Width + threadIdx.x;

        // Fetch values once to optimize memory access
        int md_row_k = Md[index_row_k];
        int md_k_col = Md[index_k_col];
        int md_row_col = Md[ROW * Width + threadIdx.x];

        // Update in-place only if condition is met
        if (md_row_col > md_row_k + md_k_col) {
            Md[ROW * Width + threadIdx.x] = md_row_k + md_k_col;
        }
    }
}