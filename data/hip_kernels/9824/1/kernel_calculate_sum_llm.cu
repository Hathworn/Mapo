#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_calculate_sum(double * dev_array_sums, unsigned int array_size, double * dev_block_sums) {

    // Declare shared memory using automatic memory allocation for the block
    extern __shared__ double shared_sum[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Efficient global to shared memory transfer
    if (i < array_size) {
        shared_sum[tid] = dev_array_sums[i];
    } else {
        shared_sum[tid] = 0.0; // Ensure to initialize with floating-point literal
    }
    __syncthreads(); // Synchronize threads

    // Use warp-level reduction when threads per block is below 32
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_sum[tid] += shared_sum[tid + s];
        }
        __syncthreads(); // Synchronize threads
    }

    // Store block result to global memory
    if (tid == 0) {
        dev_block_sums[blockIdx.x] = shared_sum[0];
    }
}