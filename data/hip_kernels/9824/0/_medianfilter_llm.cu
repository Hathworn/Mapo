#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 33 * 1024
#define threadsPerBlock 256
#define blocksPerGrid (N + threadsPerBlock - 1) / threadsPerBlock
#define RADIUS 2
// Signal/image element type
typedef int element;

// 1D MEDIAN FILTER kernel
__global__ void _medianfilter(const element* signal, element* result)
{
    __shared__ element cache[threadsPerBlock + 2 * RADIUS];
    element window[2 * RADIUS + 1]; // Adjust window size to avoid magic numbers
    int gindex = threadIdx.x + blockDim.x * blockIdx.x;
    int lindex = threadIdx.x + RADIUS;

    // Boundary check for signal array accesses
    if (gindex >= N) return;

    // Read input elements into shared memory
    cache[lindex] = signal[gindex];
    if (threadIdx.x < RADIUS)
    {
        // Prefetch border elements if within signal bounds
        if (gindex >= RADIUS)
            cache[lindex - RADIUS] = signal[gindex - RADIUS];
        if (gindex + threadsPerBlock < N)
            cache[lindex + threadsPerBlock] = signal[gindex + threadsPerBlock];
    }
    __syncthreads();

    // Window loading
    for (int j = 0; j < 2 * RADIUS + 1; ++j)
        window[j] = cache[threadIdx.x + j];

    // Orders elements (optimized bubble sort to minimize swap operations)
    for (int j = 0; j < RADIUS + 1; ++j)
    {
        for (int k = 0; k < 2 * RADIUS - j; ++k)
        {
            if (window[k] > window[k + 1])
            {
                // Swap elements
                const element temp = window[k];
                window[k] = window[k + 1];
                window[k + 1] = temp;
            }
        }
    }

    // Gets result - the middle element
    result[gindex] = window[RADIUS];
}