#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel(int32_t n, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    // Load cur_weight into a register
    float cur_weight = *cur_tot_weight;
    
    // Compute scaling value if applicable
    float val = 1.0f;
    if (max_count > 0.0f) {
        float new_scale = fmaxf(cur_weight, (float)max_count) / max_count; // Use fmaxf for performance
        val += new_scale - 1.0f; // Combine operations to reduce memory access
    }

    // Use blockDim.x and gridDim.x for efficient iteration
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = idx; i < n; i += stride) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        atomicAdd(&quadratic[diag_idx], val); // Use atomicAdd for potential concurrent access
    }

    // Ensure one thread updates linear
    if (idx == 0) {
        linear[0] += val * prior_offset;
    }
}