#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    // Use shared memory for improved performance
    extern __shared__ float shared_mem[];

    // Special case handled by the first thread
    if (thread_id == 0) {
        float real = Ar[0];
        float im = Ar[row_length];
        if (use_power) {
            shared_mem[0] = real * real;
            shared_mem[half_length] = im * im;
        } else {
            shared_mem[0] = fabs(real);
            shared_mem[half_length] = fabs(im);
        }
    }

    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {
        if (idx == 0) continue;  // Skip special case

        float2 val = reinterpret_cast<float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        if (use_power) {
            shared_mem[idx] = ret;
        } else {
            shared_mem[idx] = sqrtf(ret);
        }
    }

    // Ensure all computations are done before writing back to global memory
    __syncthreads();

    // Copy computed values from shared memory to global memory
    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {
        Aw[idx] = shared_mem[idx];
    }

    // Copy special cases from shared memory to global memory
    if (thread_id == 0) {
        Aw[0] = shared_mem[0];
        Aw[half_length] = shared_mem[half_length];
    }
}