#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Use shared memory to improve memory access pattern
    extern __shared__ float shared_feats[];
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < num_rows && j < num_cols) {
        // Load elements into shared memory
        shared_feats[threadIdx.y * blockDim.x + threadIdx.x] = feats[i * ldf + j];
        __syncthreads();

        // Use shared memory for computation to reduce global memory access
        float f = shared_feats[threadIdx.y * blockDim.x + threadIdx.x];
        feats_sq[i * lds + j] = f * f;
    }
}