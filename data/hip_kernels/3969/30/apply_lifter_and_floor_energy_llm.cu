#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    float *feats = features + frame * ldf;

    // Apply lifter coefficients using shared memory
    extern __shared__ float sh_mem[];
    if (cepstral_lifter != 0.0f) {
        for (int c = thread_id; c < num_cols; c += blockDim.x) {
            float lift = (thread_id == 0) ? lifter_coeffs[c] : 0.0f;
            __syncthreads();
            if (thread_id == 0) sh_mem[c] = lift;
            __syncthreads();
            feats[c] = feats[c] * sh_mem[c];
        }
    }

    // Use only one thread per frame to handle energy flooring
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = log(energy_floor);
        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        feats[0] = energy;
    }
}