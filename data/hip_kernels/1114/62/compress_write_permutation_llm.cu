#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compress_write_permutation(int *d_write_permutation, int *d_full_write_permutation, int *d_gcs, int total_pad_row_num, int chunk) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= total_pad_row_num) {
        return;
    }

    int chunk_id = i / chunk;
    int offset = d_gcs[chunk_id];  // Cache d_gcs[chunk_id] to reduce global memory access

    if (d_gcs[chunk_id + 1] - offset > 0) {  // Use cached offset
        int tid = i % chunk;
        // Reduce arithmetic calculation inside the write operation
        d_write_permutation[offset * chunk + tid] = d_full_write_permutation[i];
    }
}