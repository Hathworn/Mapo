#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_segmented_nnz_num(int *d_rpt, int *d_col, int *d_nnz_num, int *d_group_seg, int *d_offset, size_t seg_size, size_t seg_num, int M, int pad_M, int group_num_col)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Early return for out-of-bounds thread index
    if (i >= M) {
        return;
    }

    int offset = d_rpt[i];
    int width = d_rpt[i + 1] - d_rpt[i];

    for (int j = 0; j < width; j++) {
        int index = offset + j;
        int col = d_col[index];
        int g = col / seg_size;

        // Use atomic operations for concurrent writes
        int nnz_index = g * pad_M + i;
        int nnz_temp = atomicAdd(&d_nnz_num[nnz_index], 1);
        d_offset[index] = nnz_temp;
        d_group_seg[index] = g;
    }
}