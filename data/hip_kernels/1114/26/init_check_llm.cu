#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_check(int *d_check, int nz)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < nz) { // Combine condition check to improve warp efficiency
        d_check[i] = -1;
    }
}