#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_cl(int *nnz_num, int *cl, int chunk, int pad_M)
{
    int c_size = pad_M / chunk;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= c_size) {
        return;
    }

    int offset = chunk * i;
    int max = 0;

    // Using a single loop condition to thread the chunk elements
    for (int j = 0; j < chunk; j++) {
        int length = nnz_num[offset + j];
        max = max > length ? max : length; // Optimize using a conditional expression
    }
    cl[i] = max;
}