#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void get_c_size(int *d_c_size, int *d_full_cl, int size)
{
    // Calculate unique thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Exit if the thread index exceeds the input size
    if (i >= size) return;

    // Use fast atomic CAS increment to improve atomicAdd
    if (d_full_cl[i] != 0) {
        atomicCAS(d_c_size, atomicAdd(d_c_size, 1), *d_c_size); 
    }
}