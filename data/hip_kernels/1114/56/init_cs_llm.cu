#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_cs(int *d_cl, int *d_cs, int c_size, int chunk)
{
    // Using a single thread to initialize the first element
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        d_cs[0] = 0;
    }

    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;

    // Check bounds and compute d_cs
    if (i < c_size) {
        d_cs[i] = d_cl[i - 1] * chunk;
    }
}