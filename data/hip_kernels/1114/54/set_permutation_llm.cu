#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_permutation(int *d_permutation, int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check index within range inside bounds to reduce branching
    if (i < M) {
        d_permutation[i] = i;
    }
}