#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void zero_fill_int(int *d_array, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Avoid branching by changing the boundary check
    if (i < size) {
        d_array[i] = 0;
    }
}