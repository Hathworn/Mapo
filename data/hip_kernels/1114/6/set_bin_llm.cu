#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_bin(int *d_row_nz, int *d_bin_size, int *d_max, int M, int min, int mmin)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread ID is within range
    if (i >= M) {
        return;
    }

    // Load data from global memory
    int nz_per_row = d_row_nz[i];

    // Compute maximum using atomic operation
    atomicMax(d_max, nz_per_row);

    // Determine appropriate bin
    for (int j = 0; j < BIN_NUM - 2; j++) {
        if (nz_per_row <= (min << j)) {
            atomicAdd(d_bin_size + j + (nz_per_row > mmin), 1); // Simplified binning logic
            return;
        }
    }

    // Handle case for last bin
    atomicAdd(d_bin_size + BIN_NUM - 1, 1);
}