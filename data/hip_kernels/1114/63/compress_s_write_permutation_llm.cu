#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compress_s_write_permutation(unsigned short *d_s_write_permutation, unsigned short *d_s_write_permutation_offset, int *d_write_permutation, int c_size, int chunk)
{
    // Calculate the global index of the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (i >= c_size * chunk) {
        return;
    }

    // Calculate chunk ID for current element
    int chunk_id = i / chunk;

    // Compute s_write_permutation using modulo with USHORT_MAX
    d_s_write_permutation[i] = (unsigned short)(d_write_permutation[i] % USHORT_MAX);

    // Compute s_write_permutation_offset only for the first element in each chunk
    if (i % chunk == 0) {
        d_s_write_permutation_offset[chunk_id] = (unsigned short)(d_write_permutation[i] / USHORT_MAX);
    }
}