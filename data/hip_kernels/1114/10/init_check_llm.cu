#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_check(int *d_check, int nz)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a grid-stride loop to ensure all elements are processed
    for (; i < nz; i += blockDim.x * gridDim.x) {
        d_check[i] = -1;
    }
}