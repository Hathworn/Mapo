#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_row_perm(int *d_bin_size, int *d_bin_offset, int *d_max_row_nz, int *d_row_perm, int M, int min, int mmin)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= M) {
        return;
    }

    int nz_per_row = d_max_row_nz[i];
    int dest;

    // Loop unrolling for optimization
    if (nz_per_row <= min) {
        if (nz_per_row <= mmin) {
            dest = atomicAdd(d_bin_size, 1);
            d_row_perm[d_bin_offset[0] + dest] = i;
        }
        else {
            dest = atomicAdd(d_bin_size + 1, 1);
            d_row_perm[d_bin_offset[1] + dest] = i;
        }
        return;
    }
    if (nz_per_row <= (min << 1)) {
        if (nz_per_row <= mmin) {
            dest = atomicAdd(d_bin_size + 1, 1);
            d_row_perm[d_bin_offset[1] + dest] = i;
        }
        else {
            dest = atomicAdd(d_bin_size + 2, 1);
            d_row_perm[d_bin_offset[2] + dest] = i;
        }
        return;
    }
    // Add more unrolled conditions if needed for BIN_NUM
    // Fallback for the last bin
    dest = atomicAdd(d_bin_size + BIN_NUM - 1, 1);
    d_row_perm[d_bin_offset[BIN_NUM - 1] + dest] = i;
}