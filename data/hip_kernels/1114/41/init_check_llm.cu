#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_check(int *d_check, int nz)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use branch-less assignment within bounds
    if (i < nz) {
        d_check[i] = -1;
    }
}