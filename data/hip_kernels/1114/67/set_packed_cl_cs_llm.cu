#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_packed_cl_cs(int *d_packed_cl, int *d_packed_cs, int *d_cl, int *d_cs, int *d_gcs, int chunk_num) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Exit early if index exceeds the number of chunks
    if (i >= chunk_num) return;

    // Check if there is a valid segment to copy
    int gcs_diff = d_gcs[i + 1] - d_gcs[i];
    if (gcs_diff > 0) {
        int index = d_gcs[i];
        d_packed_cl[index] = d_cl[i];
        d_packed_cs[index] = d_cs[i];
    }
}