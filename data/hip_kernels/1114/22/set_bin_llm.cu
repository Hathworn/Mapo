#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_bin(int *d_row_nz, int *d_bin_size, int *d_max, int M, int min, int mmin)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread index is within bounds
    if (i < M) {
        int nz_per_row = d_row_nz[i];

        // Use atomic operation to find the maximum nz_per_row
        atomicMax(d_max, nz_per_row);

        // Use efficient loop for bin assignment
        for (int j = 0; j < BIN_NUM - 2; j++) {
            if (nz_per_row <= (min << j)) {
                atomicAdd(d_bin_size + (nz_per_row <= mmin ? j : j + 1), 1);
                return;
            }
        }

        // Handle case where nz_per_row falls into the last bin
        atomicAdd(d_bin_size + BIN_NUM - 1, 1);
    }
}