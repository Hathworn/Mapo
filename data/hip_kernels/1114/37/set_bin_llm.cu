#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_bin(int *d_row_nz, int *d_bin_size, int *d_max, int M, int min, int mmin)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= M) {
        return;
    }
    int nz_per_row = d_row_nz[i];

    // Use atomic to find maximum
    atomicMax(d_max, nz_per_row);

    // Optimize bin determination with loop unrolling
    if (nz_per_row <= min) {
        if (nz_per_row <= mmin) {
            atomicAdd(d_bin_size, 1);
        } else {
            atomicAdd(d_bin_size + 1, 1);
        }
    } else if (nz_per_row <= (min << 1)) {
        atomicAdd(d_bin_size + 1, 1);
    } else if (nz_per_row <= (min << 2)) {
        atomicAdd(d_bin_size + 2, 1);
    } else {
        atomicAdd(d_bin_size + BIN_NUM - 1, 1);
    }
}