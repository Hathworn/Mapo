#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_d_check_nnz(int *d_check_nnz, int *d_nnz_num, int pad_M, int SIGMA, int sigma_block_row)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= pad_M) {
        return;
    }

    // Calculate the index for d_nnz_num and d_check_nnz
    int nnz_index = blockIdx.y * pad_M + i;
    int check_index = blockIdx.y * sigma_block_row + i / SIGMA;
    
    // Avoid checking d_nnz_num repeatedly
    if (d_nnz_num[nnz_index] > 0) {
        atomicAdd(&d_check_nnz[check_index], 1);  // Perform atomic addition more efficiently
    }
}