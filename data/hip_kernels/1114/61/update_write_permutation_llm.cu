#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_write_permutation(int *write_permutation, int *nnz_num, int total_pad_row_num, int pad_M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < total_pad_row_num)  // Simplified condition to prevent out-of-bounds access
    {
        write_permutation[i] -= (i / pad_M) * pad_M;  // Calculating the reduction in a single step and applying
    }
}