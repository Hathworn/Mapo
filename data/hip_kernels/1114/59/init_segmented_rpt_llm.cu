#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_segmented_rpt(int *d_nnz_num, int *d_seg_rpt, int total_pad_row_num)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < total_pad_row_num) { // Simplified boundary condition
        // Use a single conditional statement to improve warp efficiency
        d_seg_rpt[i] = (i == 0) ? 0 : d_nnz_num[i - 1];
    }
}