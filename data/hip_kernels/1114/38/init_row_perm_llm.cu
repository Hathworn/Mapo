#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_row_perm(int *d_permutation, int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unrolling the loop for performance improvement
    if (i < M) {
        d_permutation[i] = i;
        if (i + blockDim.x < M) {
            d_permutation[i + blockDim.x] = i + blockDim.x;
            if (i + 2 * blockDim.x < M) {
                d_permutation[i + 2 * blockDim.x] = i + 2 * blockDim.x;
                if (i + 3 * blockDim.x < M) {
                    d_permutation[i + 3 * blockDim.x] = i + 3 * blockDim.x;
                }
            }
        }
    }
}