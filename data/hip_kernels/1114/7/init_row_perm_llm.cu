#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_row_perm(int *d_permutation, int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for better parallelism
    for (; i < M; i += blockDim.x * gridDim.x) {
        d_permutation[i] = i;
    }
}