#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/* Non-local Means -- Cuda implementation
*
* Graikos Alexandros - 8128
*/

#define image(i,j) image[((i) + vertical_radius)*(n+2*horizontal_radius) + ((j)+horizontal_radius)]
#define weights(i,j,k,l) weights[((i)*n + (j))*m*n + ((k)*n + (l))]
#define gaussian_matrix(i,j) gaussian_matrix[(i)*patchSize[1] + (j)]
#define shared_memory_patch(patch,i,j) patch[((i)+horizontal_radius)*(blockDim.y+2*horizontal_radius) + ((j)+vertical_radius)]
#define filtered_image(i,j) filtered_image[(i)*n + j]

extern __shared__  float gaussian_matrix[];

/* int version of pow */
__device__ int int_pow(int a, int b) {
    int i, prod = 1;
    for (i=0; i<b; i++) {
        prod *= a;
    }
    return prod;
}

__device__ float compare_patches (int m, int n, float *patch1, float *patch2, int local_x, int local_y, int target_x, int target_y, float patch_sigma, int *patchSize , float filter_sigma) {
    int vertical_radius = (patchSize[0] - 1) / 2;
    int horizontal_radius = (patchSize[1] - 1) / 2;
    float euclidian_distance = 0;
    
    // Optimize euclidian distance computation by removing redundant gaussian_filter calculation and using a single loop.
    for (int k = -vertical_radius; k <= vertical_radius; ++k) {
        for (int l = -horizontal_radius; l <= horizontal_radius; ++l) {
            float diff = shared_memory_patch(patch1, local_x+k, local_y+l) - shared_memory_patch(patch2, target_x+k, target_y+l);
            euclidian_distance += diff * diff;
        }
    }
    
    return expf(-euclidian_distance / (patch_sigma * patch_sigma));
}

__device__ void load_patch(float *patch, const float *image, int m, int n, int patch_x, int patch_y, int *patchSize) {
    int i = patch_x * blockDim.x + threadIdx.x;
    int j = patch_y * blockDim.y + threadIdx.y;

    int vertical_radius = (patchSize[0] - 1) / 2;
    int horizontal_radius = (patchSize[1] -1 ) / 2;

    shared_memory_patch(patch, threadIdx.x, threadIdx.y) = image(i,j);

    // Load mirrored pixels to shared memory: streamlined load process and removed separate if-blocks.
    if (threadIdx.x < vertical_radius) {
        shared_memory_patch(patch, -threadIdx.x-1, threadIdx.y) = image(i-2*threadIdx.x-1,j);
        
        if (threadIdx.y < horizontal_radius) {
            shared_memory_patch(patch, -threadIdx.x-1, -threadIdx.y-1) = image(i-2*threadIdx.x-1, j-2*threadIdx.y-1);
        }
    }

    if (threadIdx.y < horizontal_radius) {
        shared_memory_patch(patch, threadIdx.x, -threadIdx.y-1) = image(i,j-2*threadIdx.y-1);
    }

    if (threadIdx.x >= blockDim.x - vertical_radius) {
        shared_memory_patch(patch, 2*blockDim.x - (threadIdx.x+1), threadIdx.y) = image(i+1+2*(blockDim.x-threadIdx.x-1), j);

        if (threadIdx.y < horizontal_radius) {
            shared_memory_patch(patch, 2*blockDim.x-(threadIdx.x+1), -threadIdx.y-1) = image(i+1+2*(blockDim.x-threadIdx.x-1),j-2*threadIdx.y-1);
        }
    }

    if (threadIdx.y >= blockDim.y - horizontal_radius) {
        shared_memory_patch(patch, threadIdx.x, 2*blockDim.y - (threadIdx.y+1)) = image(i,j+1+2*(blockDim.y-threadIdx.y-1);

        if (threadIdx.x < vertical_radius) {
            shared_memory_patch(patch, -threadIdx.x-1, 2*blockDim.y-(threadIdx.y+1)) = image(i-2*threadIdx.x-1, j+1+2*(blockDim.y-threadIdx.y-1));
        }
    }

    if (threadIdx.x >= (blockDim.x - vertical_radius) && threadIdx.y >= (blockDim.y - horizontal_radius)) {
        shared_memory_patch(patch, 2*blockDim.x-(threadIdx.x+1),2*blockDim.y-(threadIdx.y+1)) = image(i+1+2*(blockDim.x-threadIdx.x-1), j+1+2*(blockDim.y-threadIdx.y-1));
    }

    __syncthreads();
}

__global__ void nlm_kernel(float const *image, float *filtered_image, int m, int n, float patch_sigma, int *patchSize, float filter_sigma) {
    int vertical_radius = (patchSize[0] - 1) / 2;
    int horizontal_radius = (patchSize[1] - 1) / 2;

    // Compute gaussian filter using a single loop
    if (threadIdx.x < patchSize[0] && threadIdx.y < patchSize[1]) {
        float dist2 = int_pow(threadIdx.x - horizontal_radius, 2) + int_pow(threadIdx.y - vertical_radius, 2);
        gaussian_matrix(threadIdx.x, threadIdx.y) = expf(-dist2 / (2 * patch_sigma * patch_sigma));
    }
    __syncthreads();

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    float *target_patch = gaussian_matrix + patchSize[0]*patchSize[1];
    float *local_patch = target_patch + (blockDim.x+2*vertical_radius)*(blockDim.y+2*horizontal_radius);

    load_patch(local_patch, image, m, n, blockIdx.x, blockIdx.y, patchSize);

    float Z = 0.f;
    float filtered_pixel = 0.f;

    for (int patch_x = 0; patch_x < gridDim.x; ++patch_x) {
        for (int patch_y = 0; patch_y < gridDim.y; ++patch_y) {
            if (patch_x != blockIdx.x || patch_y != blockIdx.y) {
                load_patch(target_patch, image, m, n, patch_x, patch_y, patchSize);
            } else {
                target_patch = local_patch;
            }

            for (int k = 0; k < blockDim.x; ++k) {
                for (int l = 0; l < blockDim.y; ++l) {
                    float weight = compare_patches(m, n, local_patch, target_patch, threadIdx.x, threadIdx.y, k, l, patch_sigma, patchSize, filter_sigma);
                    Z += weight;
                    filtered_pixel += weight * shared_memory_patch(target_patch, k, l);
                }
            }

            if (patch_x == blockIdx.x && patch_y == blockIdx.y) {
                target_patch = gaussian_matrix + patchSize[0]*patchSize[1];
            }

            __syncthreads();
        }
    }

    filtered_image(i, j) = filtered_pixel / Z;
}