#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load value from global memory
    float acc_value = *accuracy;
    // Perform division
    acc_value /= N;
    // Write back result to global memory
    *accuracy = acc_value;
}