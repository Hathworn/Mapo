#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized Monte Carlo kernel function
__global__ void gpu_monte_carlo(float *estimate, hiprandState *states) {
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int points_in_circle = 0;

    // Initialize CURAND once per thread
    hiprand_init(1234, tid, 0, &states[tid]);

    // Use local variables and unroll loop for better performance
    for (int i = 0; i < TRIALS_PER_THREAD; i++) {
        float x = hiprand_uniform(&states[tid]);
        float y = hiprand_uniform(&states[tid]);
        points_in_circle += (x * x + y * y <= 1.0f);
    }
    // Store calculated estimate
    estimate[tid] = 4.0f * points_in_circle / (float) TRIALS_PER_THREAD;
}