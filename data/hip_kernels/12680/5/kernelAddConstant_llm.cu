#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelAddConstant(int *g_a, const int b)
{
    // Calculate global index for the current thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use conditional check to prevent out-of-bounds access
    if (idx < gridDim.x * blockDim.x) {
        g_a[idx] += b;
    }
}