#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand_kernel.h"

#define TRIALS_PER_THREAD 4096
#define BLOCKS 256
#define THREADS 256

#ifdef DP
typedef double Real;
#define PI  3.14159265358979323846
#else
typedef float Real;
#define PI 3.1415926535
#endif

// Improved kernel function
__global__ void gpu_monte_carlo(Real *estimate, hiprandState *states, int trials) {
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int points_in_circle = 0;
    Real x, y;
    
    // Initialize state only once per thread instead of every trial
    hiprand_init(1234, tid, 0, &states[tid]);

    // Use shared memory to reduce global memory access latency
    extern __shared__ int shared_points_in_circle[];
    shared_points_in_circle[threadIdx.x] = 0;
    
    for(int i = 0; i < trials; i++) {
        x = hiprand_uniform(&states[tid]);
        y = hiprand_uniform(&states[tid]);
        shared_points_in_circle[threadIdx.x] += (x * x + y * y <= 1.0f);
    }

    // Aggregate results from shared memory to reduce divergence
    __syncthreads();
    if (threadIdx.x == 0) {
        int block_points_in_circle = 0;
        for (int i = 0; i < blockDim.x; i++) {
            block_points_in_circle += shared_points_in_circle[i];
        }
        estimate[blockIdx.x] = 4.0f * block_points_in_circle / (Real)(trials * blockDim.x);
    }
}