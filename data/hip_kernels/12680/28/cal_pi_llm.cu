#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cal_pi(float *sum, int nbin, float step, int nthreads, int nblocks) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;  // Sequential thread index across the blocks

    // Initialize sum for this thread
    float temp_sum = 0.0f;

    for (int i = idx; i < nbin; i += nthreads * nblocks) {
        float x = (i + 0.5f) * step;
        temp_sum += 4.0f / (1.0f + x * x);
    }

    // Store result in global sum array
    sum[idx] = temp_sum;
}