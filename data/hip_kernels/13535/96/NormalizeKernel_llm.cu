#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global thread index for accessing image data
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Boundary check to ensure we only process valid elements
    if (j < w && i < h) {
        const int pos = i * s + j;
        
        // Safely calculate the inverse scale factor
        float scale = normalization_factor[pos];
        float invScale = (scale == 0.0f) ? 1.0f : __frcp_rn(scale);  // Use reciprocal function for better performance
        
        // Apply normalization
        image[pos] *= invScale;
    }
}