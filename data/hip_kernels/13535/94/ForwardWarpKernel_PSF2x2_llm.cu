#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Use registers for frequently used variables
    float value = src[image_row_offset + j];
    
    // Precalculate corner positions and weights
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py, dx = modff(cx, &px), dy = modff(cy, &py);
    
    int tx = (int) px;
    int ty = (int) py;

    // Process each corner within limits
    if (tx < w && ty < h && tx >= 0 && ty >= 0) {
        float weight = dx * dy;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
    --tx;
    if (tx < w && ty < h && tx >= 0 && ty >= 0) {
        float weight = (1.0f - dx) * dy;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
    --ty;
    if (tx < w && ty < h && tx >= 0 && ty >= 0) {
        float weight = (1.0f - dx) * (1.0f - dy);
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
    ++tx;
    if (tx < w && ty < h && tx >= 0 && ty >= 0) {
        float weight = dx * (1.0f - dy);
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}