#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate global thread index
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index does not exceed the number of elements
    if (ptidx >= npoints) return;

    // Load the location data
    short2 loc = loc_[ptidx];

    // Perform the scaled assignment
    x[ptidx] = loc.x * scale;
    y[ptidx] = loc.y * scale;
}