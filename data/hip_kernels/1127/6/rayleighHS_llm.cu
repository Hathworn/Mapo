#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rayleighHS(double *Mh_d, double *pressure_d, double *Rho_d, double *Altitude_d, double surf_drag, double bl_sigma, double Gravit, double time_step, int num) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int nv = gridDim.y;
    int lev = blockIdx.y;

    if (id < num) {
        double sigma;
        double sigmab = bl_sigma;
        double kf = surf_drag;
        double kv_hs;
        double ps, pre;
        double psm1;

        // Calculate surface pressure
        psm1 = __fma_rn(-Rho_d[id * nv + 0], Gravit, pressure_d[id * nv + 1] - (Altitude_d[0] + Altitude_d[1]));
        ps = 0.5 * (pressure_d[id * nv + 0] + psm1);

        pre = pressure_d[id * nv + lev];
        // Use fast reciprocal operation for sigma as pre / ps
        sigma = __frcp_rn(ps) * pre;

        // Momentum dissipation constant using max intrinsic
        kv_hs = __dmul_rn(kf, fmax(0.0, (sigma - sigmab) / (1.0 - sigmab)));

        // Update momenta using loop unrolling
        #pragma unroll
        for (int k = 0; k < 3; ++k) {
            Mh_d[id * 3 * nv + lev * 3 + k] /= (1.0 + __dmul_rn(kv_hs, time_step));
        }
    }
}