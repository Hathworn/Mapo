#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalcAngMom(double *AngMomx_d, double *AngMomy_d, double *AngMomz_d, double *GlobalAMx_d, double *GlobalAMy_d, double *GlobalAMz_d, double *Mh_d, double *Rho_d, double  A, double  Omega, double *Altitude_d, double *Altitudeh_d, double *lonlat_d, double *areasT, double *func_r_d, int     num, bool    DeepModel) {

    int id  = blockIdx.x * blockDim.x + threadIdx.x;
    int nv  = gridDim.y;
    int lev = blockIdx.y;

    if (id < num) {
        double AMx, AMy, AMz;
        double rx, ry, rz, r;

        // Calculate control volume
        double zup, zlow, Vol;
        zup  = Altitudeh_d[lev + 1] + A;
        zlow = Altitudeh_d[lev] + A;
        if (DeepModel) {
            Vol = areasT[id] / pow(A, 2) * (pow(zup, 3) - pow(zlow, 3)) / 3;
        } else {
            Vol = areasT[id] * (zup - zlow);
        }

        // Precompute cosines and sines to avoid recalculating
        double cos_lat = cos(lonlat_d[id * 2 + 1]);
        double cos_lon = cos(lonlat_d[id * 2]);
        double sin_lon = sin(lonlat_d[id * 2]);

        // Radius vector
        r  = A + Altitude_d[lev];
        rx = r * func_r_d[id * 3 + 0];
        ry = r * func_r_d[id * 3 + 1];
        rz = r * func_r_d[id * 3 + 2];

        int index = id * 3 * nv + lev * 3;
        int rho_index = id * nv + lev;

        // Angular momentum r x p
        AMx = ry * Mh_d[index + 2] - rz * Mh_d[index + 1]
            - Rho_d[rho_index] * Omega * r * rz * cos_lat * cos_lon;
        AMy = -rx * Mh_d[index + 2] + rz * Mh_d[index + 0]
            - Rho_d[rho_index] * Omega * r * rz * cos_lat * sin_lon;
        AMz = rx * Mh_d[index + 1] - ry * Mh_d[index + 0]
            + Rho_d[rho_index] * Omega * r * r * cos_lat * cos_lat;

        // Total in control volume
        AngMomx_d[id * nv + lev] = AMx * Vol;
        AngMomy_d[id * nv + lev] = AMy * Vol;
        AngMomz_d[id * nv + lev] = AMz * Vol;
    }
}