#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalcEntropy(double *Entropy_d, double *pressure_d, double *temperature_d, double Cp, double Rd, double A, double P_Ref, double *Altitude_d, double *Altitudeh_d, double *lonlat_d, double *areasT, double *func_r_d, int num, bool DeepModel) {

    int id  = blockIdx.x * blockDim.x + threadIdx.x;
    int nv  = gridDim.y;
    int lev = blockIdx.y;

    if (id < num) {
        // Precompute values used more than once
        double kappa = Rd / Cp;
        double pressure_inv = pow(P_Ref / pressure_d[id * nv + lev], kappa);
        double potT  = temperature_d[id * nv + lev] * pressure_inv;
        double Sdens = Cp * log(potT);

        // Calculate control volume
        double zup = Altitudeh_d[lev + 1] + A;
        double zlow = Altitudeh_d[lev] + A;
        double Vol;
        if (DeepModel) {
            double zup3 = pow(zup, 3);
            double zlow3 = pow(zlow, 3);
            Vol = areasT[id] / pow(A, 2) * (zup3 - zlow3) / 3;
        }
        else {
            Vol = areasT[id] * (zup - zlow);
        }

        // Total energy in the control volume
        Entropy_d[id * nv + lev] = Sdens * Vol;
    }
}