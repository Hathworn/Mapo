#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void isnan_check_device(double *array, int size, bool *check) {
    // Calculate the unique thread index 
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    // Use shared memory to improve performance by reducing global memory access
    __shared__ bool local_check;

    // Initialize shared memory for each block
    if (threadIdx.x == 0) local_check = false;
    __syncthreads();

    // Check for NaN only if within bounds
    if (idx < size && ::isnan(array[idx])) {
        atomicExch(&local_check, true); // Use atomic operation to prevent race conditions
    }

    // Sync to ensure all threads have updated local_check
    __syncthreads();

    // Update global 'check' from local_check
    if (threadIdx.x == 0 && local_check) {
        atomicExch(check, true); // Atomically update global check
    }
}