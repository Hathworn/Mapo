#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_mean(double* pressure_mean_d, double* pressure_d, double* Rho_mean_d, double* Rho_d, double* Mh_mean_d, double* Mh_d, double* Wh_mean_d, double* Wh_d, int n_since_out, int num) {

    int id  = blockIdx.x * blockDim.x + threadIdx.x;
    int nv  = gridDim.y;
    int lev = blockIdx.y;

    if (id < num) {
        double factor = 1.0 / n_since_out;  // Reuse common factor

        int base_index = id * nv + lev;
        pressure_mean_d[base_index] = factor * (pressure_mean_d[base_index] * (n_since_out - 1) + pressure_d[base_index]);
        Rho_mean_d[base_index] = factor * (Rho_mean_d[base_index] * (n_since_out - 1) + Rho_d[base_index]);

        int mh_base_index = 3 * id * nv + 3 * lev;
        Mh_mean_d[mh_base_index] = factor * (Mh_mean_d[mh_base_index] * (n_since_out - 1) + Mh_d[mh_base_index]);
        Mh_mean_d[mh_base_index + 1] = factor * (Mh_mean_d[mh_base_index + 1] * (n_since_out - 1) + Mh_d[mh_base_index + 1]);
        Mh_mean_d[mh_base_index + 2] = factor * (Mh_mean_d[mh_base_index + 2] * (n_since_out - 1) + Mh_d[mh_base_index + 2]);

        int wh_base_index = id * (nv + 1) + lev;
        Wh_mean_d[wh_base_index] = factor * (Wh_mean_d[wh_base_index] * (n_since_out - 1) + Wh_d[wh_base_index]);

        // Ensure last level update for Wh_mean_d
        if (lev == nv - 1) {
            Wh_mean_d[wh_base_index + 1] = factor * (Wh_mean_d[wh_base_index + 1] * (n_since_out - 1) + Wh_d[wh_base_index + 1]);
        }
    }
}