#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CalcMass(double *Mass_d, double *GlobalMass_d, double *Rho_d, double A, double *Altitudeh_d, double *lonlat_d, double *areasT, int num, bool DeepModel) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int nv = gridDim.y;
    int lev = blockIdx.y;

    if (id < num) {
        // Pre-calculate and store repeated terms
        double zup = Altitudeh_d[lev + 1] + A;
        double zlow = Altitudeh_d[lev] + A;
        double deltaZ = zup - zlow;

        // Use ternary operator for concise conditional expression
        double Vol = DeepModel ?
                     (areasT[id] / (A * A) * (zup * zup * zup - zlow * zlow * zlow) / 3) :
                     (areasT[id] * deltaZ);

        // Calculate mass
        Mass_d[id * nv + lev] = Rho_d[id * nv + lev] * Vol;
    }
}