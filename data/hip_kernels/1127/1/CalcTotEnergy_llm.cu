#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalcTotEnergy(double *Etotal_d, double *GlobalE_d, double *Mh_d, double *W_d, double *Rho_d, double *temperature_d, double  Gravit, double  Cp, double  Rd, double  A, double *Altitude_d, double *Altitudeh_d, double *lonlat_d, double *areasT, double *func_r_d, int     num, bool    DeepModel) {

    int id  = blockIdx.x * blockDim.x + threadIdx.x;
    int nv  = gridDim.y;
    int lev = blockIdx.y;

    if (id < num) {
        double Ek, Eint, Eg;
        double wx, wy, wz;
        double Cv = Cp - Rd;
        double zup, zlow, Vol;

        // Pre-calculate common terms to reduce repeated calculations
        double lon_cos = cos(lonlat_d[id * 2 + 1]);
        double lon_sin = sin(lonlat_d[id * 2 + 1]);
        double lat_cos = cos(lonlat_d[id * 2]);
        double lat_sin = sin(lonlat_d[id * 2]);

        zup  = Altitudeh_d[lev + 1] + A;
        zlow = Altitudeh_d[lev] + A;

        // Optimize volume computation
        Vol = areasT[id] * (DeepModel ? ((pow(zup, 3) - pow(zlow, 3)) / (3 * pow(A, 2))) : (zup - zlow));

        // Optimize wind calculations using precomputed trigonometric values
        wx = W_d[id * nv + lev] * lon_cos * lat_cos;
        wy = W_d[id * nv + lev] * lon_cos * lat_sin;
        wz = W_d[id * nv + lev] * lon_sin;

        // Optimize energy calculations
        double rho = Rho_d[id * nv + lev];
        Ek = 0.5 * ((Mh_d[id * 3 * nv + lev * 3 + 0] + wx) * (Mh_d[id * 3 * nv + lev * 3 + 0] + wx)
                  + (Mh_d[id * 3 * nv + lev * 3 + 1] + wy) * (Mh_d[id * 3 * nv + lev * 3 + 1] + wy)
                  + (Mh_d[id * 3 * nv + lev * 3 + 2] + wz) * (Mh_d[id * 3 * nv + lev * 3 + 2] + wz)) / rho;

        Eint = Cv * temperature_d[id * nv + lev] * rho;
        Eg   = rho * Gravit * Altitude_d[lev];

        // Store total energy
        Etotal_d[id * nv + lev] = (Ek + Eint + Eg) * Vol;
    }
}