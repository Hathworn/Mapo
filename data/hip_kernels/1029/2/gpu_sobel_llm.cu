#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE  16
#define HEADER_SIZE 122
#define BLOCK_SIZE_SH 18

typedef unsigned char BYTE;

/**
* Structure that represents a BMP image.
*/
typedef struct
{
    int   width;
    int   height;
    float *data;
} BMPImage;

typedef struct timeval tval;

BYTE g_info[HEADER_SIZE]; // Reference header

/**
* Reads a BMP 24bpp file and returns a BMPImage structure.
* Thanks to https://stackoverflow.com/a/9296467
*/
__device__ float gpu_applyFilter(float *image, int stride, float *matrix, int filter_dim)
{
    float pixel = 0.0f;
    for (int h = 0; h < filter_dim; h++)
    {
        int offset = h * stride;
        int offset_kernel = h * filter_dim;
        for (int w = 0; w < filter_dim; w++)
        {
            pixel += image[offset + w] * matrix[offset_kernel + w];
        }
    }
    return pixel;
}

__global__ void gpu_sobel(int width, int height, float *image, float *image_out)
{
    __shared__ float sh_block[BLOCK_SIZE_SH * BLOCK_SIZE_SH];

    // Load Sobel kernels into shared memory
    __shared__ float sobel_x[9];
    __shared__ float sobel_y[9];
    if (threadIdx.x < 9) {
        sobel_x[threadIdx.x] = (threadIdx.x == 0 || threadIdx.x == 6) ? 1.0f : (threadIdx.x == 1 || threadIdx.x == 7) ? 0.0f : (threadIdx.x == 2 || threadIdx.x == 8) ? -1.0f : (threadIdx.x == 3 || threadIdx.x == 5) ? 2.0f : 0.0f;
        sobel_y[threadIdx.x] = (threadIdx.x == 0 || threadIdx.x == 2) ? 1.0f : (threadIdx.x == 3 || threadIdx.x == 5) ? 2.0f : (threadIdx.x == 6 || threadIdx.x == 8) ? -1.0f : 0.0f;
    }
    __syncthreads();

    // Calculate global indices
    const int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (index_x < (width - 2) && index_y < (height - 2))
    {
        int offset_t = index_y * width + index_x;
        int offset   = (index_y + 1) * width + (index_x + 1);
        int offset_shared = threadIdx.y * BLOCK_SIZE_SH + threadIdx.x;

        // Load data to shared memory
        sh_block[offset_shared] = image[offset_t];
        
        // Handle boundary conditions more efficiently
        if (threadIdx.y < 3 && index_y < height - 3) {
            sh_block[offset_shared + BLOCK_SIZE_SH] = image[offset_t + width];
            sh_block[offset_shared + BLOCK_SIZE_SH * 2] = image[offset_t + 2 * width];
        }
        if (threadIdx.x < 3 && index_x < width - 3) {
            sh_block[offset_shared + 1] = image[offset_t + 1];
            sh_block[offset_shared + 2] = image[offset_t + 2];
        }
        
        __syncthreads();

        float gx = gpu_applyFilter(&sh_block[offset_shared], BLOCK_SIZE_SH, sobel_x, 3);
        float gy = gpu_applyFilter(&sh_block[offset_shared], BLOCK_SIZE_SH, sobel_y, 3);

        // Compute the magnitude and store it
        image_out[offset] = sqrtf(gx * gx + gy * gy);
    }
}