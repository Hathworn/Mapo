#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE  16
#define HEADER_SIZE 122
#define BLOCK_SIZE_SH 18

typedef unsigned char BYTE;

/**
* Structure that represents a BMP image.
*/
typedef struct
{
    int   width;
    int   height;
    float *data;
} BMPImage;

typedef struct timeval tval;

BYTE g_info[HEADER_SIZE]; // Reference header

__global__ void gpu_grayscale(int width, int height, float *image, float *image_out)
{
    // Calculate pixel index
    const int h = blockIdx.y * blockDim.y + threadIdx.y;
    const int w = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds
    if (h < height && w < width)
    {
        // Shared memory for pixel data to reduce global memory access
        __shared__ float pixel_block[BLOCK_SIZE][BLOCK_SIZE][3];

        int imageOffset = (h * width + w) * 3;
        int blockOffset = threadIdx.y * blockDim.x + threadIdx.x;

        // Load pixel data into shared memory
        for (int i = 0; i < 3; i++)
        {
            pixel_block[threadIdx.y][threadIdx.x][i] = image[imageOffset + i];
        }
        
        __syncthreads(); // Synchronize threads within block

        // Compute grayscale in shared memory
        image_out[h * width + w] = pixel_block[threadIdx.y][threadIdx.x][0] * 0.0722f + // B
                                   pixel_block[threadIdx.y][threadIdx.x][1] * 0.7152f + // G
                                   pixel_block[threadIdx.y][threadIdx.x][2] * 0.2126f;  // R
    }
}