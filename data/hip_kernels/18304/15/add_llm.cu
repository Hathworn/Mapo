#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(float *array_a, float *array_b, float *array_c, int size) {
    // Optimize tid and step calculation using variables and ensure better memory access
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    
    // Use unrolling for better performance
    for (int i = tid; i < size; i += step) {
        array_c[i] = array_a[i] + array_b[i];
    }
}