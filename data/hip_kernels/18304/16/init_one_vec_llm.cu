#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_one_vec(float *d_one_vec, size_t length) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop with stride to ensure all elements are set across multiple threads
    for (; i < length; i += blockDim.x * gridDim.x) {
        d_one_vec[i] = 1.f; // Initialize element to 1.0
    }
}