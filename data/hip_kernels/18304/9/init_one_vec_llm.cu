#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_one_vec(float *d_one_vec, size_t length) {
    // Calculate the global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread is within bounds
    if (i < length) {
        // Set the value to 1.0f
        d_one_vec[i] = 1.0f;
    }
}