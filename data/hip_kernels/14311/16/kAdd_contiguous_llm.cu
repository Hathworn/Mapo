#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Improved index calculation for warp divergence reduction
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < numEls) {
        dest[idx] = a[idx] + b[idx];
    }
    
    // Additional loop to cover any remaining elements if numEls isn't a multiple of numThreads
    idx += blockDim.x * gridDim.x;
    while (idx < numEls) {
        dest[idx] = a[idx] + b[idx];
        idx += blockDim.x * gridDim.x;
    }
}