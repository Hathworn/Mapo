#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refine_dilateFPPlaneDepthMapXpYp_kernel(float* fpPlaneDepthMap, int fpPlaneDepthMap_p, float* maskMap, int maskMap_p, int width, int height, int xp, int yp, float fpPlaneDepth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds only once for improved performance
    if((x < width) && (y < height))
    {
        int new_x = x + xp;
        int new_y = y + yp;

        // Simplified bounds checking for the target position
        if((new_x >= 0) && (new_y >= 0) && (new_x < width) && (new_y < height))
        {
            float depth = maskMap[y * maskMap_p + x];
            if(depth > 0.0f)
            {
                fpPlaneDepthMap[new_y * fpPlaneDepthMap_p + new_x] = fpPlaneDepth;
            }
        }
    }
}