#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generateGaussian_kernel(float* og, float delta, int radius)
{
    int x = threadIdx.x - radius;
    float deltaSquared = 2 * delta * delta;  // Precompute constant term
    og[threadIdx.x] = __expf(-(x * x) / deltaSquared);  // Use precomputed term
}