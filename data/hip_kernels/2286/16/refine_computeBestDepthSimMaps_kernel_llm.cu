#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float refineDepthSubPixel(const float3& depths, const float3& sims)
{
    float outDepth = -1.0f;

    float simM1 = (sims.x + 1.0f) / 2.0f;
    float simP1 = (sims.z + 1.0f) / 2.0f;
    float sim1 = (sims.y + 1.0f) / 2.0f;

    if((simM1 > sim1) && (simP1 > sim1))
    {
        float dispStep = -((simP1 - simM1) / (2.0f * (simP1 + simM1 - 2.0f * sim1)));

        float floatDepthM1 = depths.x;
        float floatDepthP1 = depths.z;

        float b = (floatDepthP1 + floatDepthM1) / 2.0f;
        float a = b - floatDepthM1;

        outDepth = a * dispStep + b;
    }

    return outDepth;
}

__global__ void refine_computeBestDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float3* isims, int isims_p, float3* idpts, int idpts_p, int width, int height, float simThr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if((x < width) && (y < height))
    {
        float3 depths = idpts[y * idpts_p + x];
        float3 sims = isims[y * isims_p + x];

        // Use ternary operators efficiently for compactness
        float outDepth = (sims.x < sims.y) ? depths.x : depths.y;
        float outSim = (sims.x < sims.y) ? sims.x : sims.y;
        outDepth = (sims.z < outSim) ? depths.z : outDepth;
        outSim = (sims.z < outSim) ? sims.z : outSim;

        float refinedDepth = refineDepthSubPixel(depths, sims);
        if(refinedDepth > 0.0f)
        {
            outDepth = refinedDepth;
        }

        // Simplified conditional assignment
        bool belowThreshold = outSim < simThr;
        osim[y * osim_p + x] = belowThreshold ? outSim : 1.0f;
        odpt[y * odpt_p + x] = belowThreshold ? outDepth : -1.0f;
    }
}