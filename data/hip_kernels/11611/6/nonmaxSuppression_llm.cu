#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int tex_i(const int * ptData, int y, int x, int step)
{
    return ptData[y * step + x];
}

__global__ void nonmaxSuppression(const short2* kpLoc_Device, int count, const int* score_DeviceMat, int cols, int rows, short2* locFinal, float* responseFinal)
{
    const int kpIdx = threadIdx.x + blockIdx.x * blockDim.x;

    if (kpIdx < count)
    {
        short2 loc = kpLoc_Device[kpIdx];
        int score_Device = tex_i(score_DeviceMat, loc.y, loc.x, cols);

        // Check the boundary conditions to avoid out-of-bounds memory access
        bool withinBounds = (loc.y > 0 && loc.y < rows - 1 && loc.x > 0 && loc.x < cols - 1);

        if (withinBounds) 
        {
            // Check if current point is a local maximum
            bool ismax = score_Device > tex_i(score_DeviceMat, loc.y - 1, loc.x - 1, cols) &&
                         score_Device > tex_i(score_DeviceMat, loc.y - 1, loc.x, cols) &&
                         score_Device > tex_i(score_DeviceMat, loc.y - 1, loc.x + 1, cols) &&
                         score_Device > tex_i(score_DeviceMat, loc.y, loc.x - 1, cols) &&
                         score_Device > tex_i(score_DeviceMat, loc.y, loc.x + 1, cols) &&
                         score_Device > tex_i(score_DeviceMat, loc.y + 1, loc.x - 1, cols) &&
                         score_Device > tex_i(score_DeviceMat, loc.y + 1, loc.x, cols) &&
                         score_Device > tex_i(score_DeviceMat, loc.y + 1, loc.x + 1, cols);

            if (ismax)
            {
                const unsigned int ind = atomicInc(&g_counter, (unsigned int)(-1));
                locFinal[ind] = loc;
                responseFinal[ind] = static_cast<float>(score_Device);
            }
        }
    }
}