#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PyrDown_y_g(u_int8_t *ptGrayIn,u_int8_t *ptGrayOut, int w, int h)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < w && iy < h)
    {
        // Load all the required pixels in advance to avoid multiple global memory accesses
        int base = ix * 2;
        int offset = iy * 2 * w * 2;

        float p_2 = ptGrayIn[base + (iy * 2 - 2) * w * 2] / 16.0f;
        float p_1 = ptGrayIn[base + (iy * 2 - 1) * w * 2] / 4.0f;
        float p0 = 3.0f * ptGrayIn[base + iy * 2 * w * 2] / 8.0f;
        float pp1 = ptGrayIn[base + (iy * 2 + 1) * w * 2] / 4.0f;
        float pp2 = ptGrayIn[base + (iy * 2 + 2) * w * 2] / 16.0f;

        int output = p_2 + p_1 + p0 + pp1 + pp2;
        ptGrayOut[ix + iy * w] = min(output, 255);
    }
}