#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Each thread gets a unique seed, different sequence number
    hiprand_init(seed, tidx, tidx, &state[tidx]);
}