#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    // Ensure thread index is within bounds
    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        // Use fast math intrinsic for logarithm
        labelLogProbs[tx] = __logf(labelp);

        // Compute correct probabilities efficiently
        if (labelp != maxp) {
            correctProbs[tx] = 0.0f;
        } else {
            int numMax = 0;
            // Use shared memory to reduce frequent global memory access (if beneficial)
            for (int i = 0; i < numOut; i++) {
                numMax += __syncthreads_or(probs[i * numCases + tx] == maxp);
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}