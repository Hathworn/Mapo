#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel to prevent race conditions and improve performance
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use atomic operation for division when multiple threads access shared memory
    atomicExch(accuracy, *accuracy / N);
}