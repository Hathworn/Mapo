#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;
    
    // Use unrolled loop to enhance memory coalescing and reduce loop overhead
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();
    
    // Optimize reduction with a parallel tree-reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }
    
    // Compute gradient using the reduction result
    if (idx == 0) {
        tmp = reduction_buffer[0];
    }
    __syncthreads();
    
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}