#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DP_BLOCKSIZE 512  // Define block size for dot product

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0;  // Initialize sum for each thread

    // Unrolled iteration to increase performance
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        sum += a[eidx] * b[eidx];
    }

    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Optimize reduction using loop unrolling for better performance
    if (threadIdx.x < 256) shmem[threadIdx.x] += shmem[threadIdx.x + 256];
    __syncthreads();

    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    __syncthreads();

    if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    __syncthreads();

    // Use a single warp for final reduction steps
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;  // Use volatile to prevent reordering
        *mysh += mysh[threadIdx.x + 32];
        *mysh += mysh[threadIdx.x + 16];
        *mysh += mysh[threadIdx.x + 8];
        *mysh += mysh[threadIdx.x + 4];
        *mysh += mysh[threadIdx.x + 2];
        *mysh += mysh[threadIdx.x + 1];
    }

    // Save result from thread 0
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}