#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Early return if pointer exceeds boundary
    if (ptr >= (unsigned long*) end_ptr) {
        return;
    }

    // Use threadIdx.x for offset computation to enhance parallelism
    unsigned int offset = threadIdx.x * sizeof(unsigned long);
    if (offset < BLOCKSIZE) {
        ptr[threadIdx.x] = (unsigned long) &ptr[threadIdx.x];
    }

    return;
}
```
