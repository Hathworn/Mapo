#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    if (ptr >= (unsigned int*) end_ptr) {
        return; // Exit if pointer is out of bounds
    }

    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x){
        // Use all threads in the block for parallel writes
        ptr[i] = pattern;
    }

    __syncthreads(); // Ensure all writes are completed
}