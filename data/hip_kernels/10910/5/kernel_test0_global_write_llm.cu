#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    // Use thread indexing
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int* orig_ptr = ptr;
    
    unsigned int pattern = 1 << tid; // Each thread gets a different pattern
    unsigned long mask = 4 << tid;   // Each thread gets a different mask
    
    ptr = (unsigned int*)((unsigned long)orig_ptr | mask);
    if (ptr >= end_ptr) return;
    
    *ptr = pattern;
    
    while(true) {
        mask = mask << 1;
        ptr = (unsigned int*)((unsigned long)orig_ptr | mask);
        if (ptr >= end_ptr) break;
        *ptr = pattern;
        pattern = pattern << 1;
    }
}