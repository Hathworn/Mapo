#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* start_ptr = (unsigned int*) _start_ptr;
    
    // Early exit if ptr is out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Using threadIdx.x to spread the loop work across threads in a warp
    unsigned int idx = threadIdx.x;
    for (; idx < BLOCKSIZE / sizeof(unsigned int); idx += blockDim.x) {
        ptr[idx] = start_ptr[idx];
    }
}