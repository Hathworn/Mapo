#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    unsigned int i;
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Optimize memory access by unrolling and reducing branching
    unsigned int limit = BLOCKSIZE / sizeof(unsigned int);

    for (i = offset; i < limit; i += MOD_SZ) {
        ptr[i] = p1;
    }

    for (i = 0; i < limit; ++i) {
        if (i % MOD_SZ != offset) {
            ptr[i] = p2;
        }
    }

    return;
}