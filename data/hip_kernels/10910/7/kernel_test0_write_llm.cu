#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    // Calculate the starting pointer for each block
    unsigned int* orig_ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* ptr = orig_ptr;
    
    // Check bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);
    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Write initial pattern
    *ptr = pattern;

    // Efficient loop with bit manipulation
    while (ptr < block_end) {
        ptr = (unsigned int*) (((unsigned long)orig_ptr) | mask);
        
        if (ptr == orig_ptr) {
            mask <<= 1;
            continue;
        }
        
        if (ptr >= block_end) {
            break;
        }

        *ptr = pattern;

        pattern <<= 1;
        mask <<= 1;
    }
    return;
}