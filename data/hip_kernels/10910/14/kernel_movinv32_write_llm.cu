#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check if the starting pointer is out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int k = offset;
    unsigned int pat = pattern;

    // Use a single loop with memory coalescing optimization
    int i = threadIdx.x;
    int stride = blockDim.x;

    for (; i < BLOCKSIZE / sizeof(unsigned int); i += stride) {
        ptr[i] = pat;
        k++;
        // Calculate the new pattern value conditionally
        if (k >= 32) {
            k = 0;
            pat = lb;
        } else {
            pat = (pat << 1) | sval;
        }
    }
}