#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Return early if out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    const unsigned int half_count = BLOCKSIZE / sizeof(unsigned int) / 2;
    unsigned int* ptr_mid = ptr + half_count;

    // Use a single loop with register variables to reduce memory accesses
    for (unsigned int i = 0; i < half_count; i++) {
        unsigned int temp = ptr[i];
        if (i < half_count - 8) {
            ptr[i + 8] = temp;
        }
        if (i < 8) {
            ptr[i] = ptr_mid[half_count - 8 + i];
        }
        ptr_mid[i] = temp;
    }
}