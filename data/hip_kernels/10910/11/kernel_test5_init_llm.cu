#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_init(char* _ptr, char* end_ptr)
{
    // Improved memory alignment and coalescing for better performance
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Use threadIdx for better parallelism
    unsigned int p1 = 1;
    for (int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x * 16) {
        for (int j = threadIdx.x; j < 16; j += blockDim.x) {
            unsigned int p2 = ~p1;
            ptr[i + j] = ((j & 1) == 0) ? p1 : p2;
        }
        
        __syncthreads();  // Synchronize threads within block

        if (threadIdx.x == 0) { // Only thread 0 updates p1 to reduce redundant calculations
            p1 <<= 1;
            if (p1 == 0) {
                p1 = 1;
            }
        }
        
        __syncthreads();  // Ensure p1 update is visible to all threads
    }
}