#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    // Calculate global thread index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate pointer for this thread
    unsigned int* ptr = (unsigned int*) (_ptr + idx * sizeof(unsigned int));
    
    // Ensure pointer is within bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }
    
    // Write pattern to memory
    *ptr = pattern;
}