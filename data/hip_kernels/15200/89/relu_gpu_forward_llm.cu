#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Use grid-stride loop for better scalability across multiple GPUs
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = tid; i < N; i += stride) {
        out[i] = in[i] > 0 ? in[i] : 0;
    }
}