#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Vectorized computation 
    if (tid < N) {
        float data = indata[tid];
        float grad = outgrad[tid];
        ingrad[tid] = data > 0 ? grad : 0;
    }
}