#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];

    // Optimize memory access by using shared variables for weight computations
    float weights[] = {dx * dy, (1.0f - dx) * dy, (1.0f - dx) * (1.0f - dy), dx * (1.0f - dy)};
    int offsets[][2] = {{0, 0}, {-1, 0}, {-1, -1}, {0, -1}};

    for (int k = 0; k < 4; ++k) {
        int x = tx + offsets[k][0];
        int y = ty + offsets[k][1];
        if (!(x >= w || x < 0 || y >= h || y < 0)) {
            float weight = weights[k];
            atomicAdd(dst + y * image_stride + x, value * weight);
            atomicAdd(normalization_factor + y * image_stride + x, weight);
        }
    }
}