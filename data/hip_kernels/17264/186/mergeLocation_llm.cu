#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (ptidx < npoints)
    {
        // Load loc into registers for faster access
        short2 loc = loc_[ptidx];

        // Pre-compute scaled values for x and y
        float scaledX = loc.x * scale;
        float scaledY = loc.y * scale;

        // Write computed values to global memory
        x[ptidx] = scaledX;
        y[ptidx] = scaledY;
    }
}