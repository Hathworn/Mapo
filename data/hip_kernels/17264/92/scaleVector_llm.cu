#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len) 
{
    // Calculate global index using built-in functions for better readability
    const int pos = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;

    // Use early return for out-of-bounds threads
    if (pos < len) 
    {
        d_res[pos] = d_src[pos] * scale; // Scale vector element-wise
    }
}