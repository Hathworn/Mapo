#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Calculate global thread indices for x and y directions
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Ensure the thread is within bounds
    if (i >= h || j >= w) return;

    // Pre-compute row offsets for efficiency
    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Read flow data
    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Compute target coordinates
    // Simplify corner calculations by using implicit casting
    float cx = u_ * time_scale + j + 1.0f;
    float cy = v_ * time_scale + i + 1.0f;
    
    // Calculate rounded target pixel location
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Read from source image
    float value = src[image_row_offset + j];

    // Atomic addition to avoid race conditions in updating dst
    // Check if the target pixel is within valid bounds
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}