#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image) {
    // Calculate linear thread index from 2D block and grid dimensions
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Early exit for threads outside the boundaries
    if (idx >= w || idy >= h) return;

    // Compute the global linear index
    int pos = idy * w + idx;

    // Set the image value
    image[pos] = value;
}