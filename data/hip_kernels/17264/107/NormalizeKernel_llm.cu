#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global index once to avoid repetitive computation
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Check bounds
    if (i < h && j < w) {
        // Calculate position index
        const int pos = i * s + j;
        
        // Read normalization factor
        float scale = normalization_factor[pos];

        // Avoid dividing by zero
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

        // Scale image value
        image[pos] *= invScale;
    }
}