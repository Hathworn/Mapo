#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generateImg(unsigned char *data, unsigned char *img, unsigned char *tabDepth, int4 *_tabParents, int i, int tailleTab) {
    int thx = blockIdx.x * blockDim.x + threadIdx.x;
    int thy = blockIdx.y * blockDim.y + threadIdx.y;
    int ThId = thy * tailleTab + thx;
    
    // Early exit if condition not met (avoid unnecessary calculations)
    if (data[ThId] != 0 || tabDepth[ThId] != i || i == 1) {
        return;
    }

    // Reduce memory accesses by using temporary variables
    int4 parents = _tabParents[ThId];
    int nbPar = 0;
    int dataSum = 0;
    
    if (parents.x != -1) { nbPar++; dataSum += data[parents.x]; }
    if (parents.y != -1) { nbPar++; dataSum += data[parents.y]; }
    if (parents.z != -1) { nbPar++; dataSum += data[parents.z]; }
    if (parents.w != -1) { nbPar++; dataSum += data[parents.w]; }
    
    // Ensure nbPar is not zero before division
    if (nbPar > 0) {
        data[ThId] = dataSum / nbPar;
    }
    
    img[ThId] = data[ThId];
}