#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_monte_carlo(float *estimate, hiprandState *states) {
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int total_threads = gridDim.x * blockDim.x;
    int points_in_circle = 0;
    float x, y;

    // Initialize once per thread block instead of per thread
    hiprand_init(1234, tid, 0, &states[tid]);

    // Use a stride loop for better memory coalescing and resource utilization
    for (unsigned int i = tid; i < TRIALS_PER_THREAD * total_threads; i += total_threads) {
        x = hiprand_uniform(&states[tid]);
        y = hiprand_uniform(&states[tid]);
        points_in_circle += (x * x + y * y <= 1.0f); // count if x & y is in the circle.
    }

    estimate[tid] = 4.0f * points_in_circle / (float)TRIALS_PER_THREAD; // return estimate of pi
}