#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuImageBrighten(const float *dev_image, float *dev_out, int w, int h)
{
    // Use blockDim and gridDim for scalability
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Condition to handle image bounds and avoid out of bounds memory access
    if (x < w && y < h) {
        int pos = x + w * y;
        dev_out[pos] = min(255.0f, dev_image[pos] + 50.0f);
    }
}