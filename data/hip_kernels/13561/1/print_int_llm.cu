#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Using shared memory for better performance
__global__ void print_int(int* x, int leng) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ int shared_x[];

    // Load data into shared memory
    if(i < leng) {
        shared_x[threadIdx.x] = x[i];
        __syncthreads();

        // Print from shared memory
        printf("%d,", shared_x[threadIdx.x]);
    }
}