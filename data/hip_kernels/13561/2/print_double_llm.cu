#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_double(double* x, int leng) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure thread index is within bounds and streamline cache access
    if(i < leng) {
        double value = x[i]; // Minimize memory traffic by caching value
        printf("%lf,", value); // Use cached value instead of direct access
    }
}