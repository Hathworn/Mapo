#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void print_float(float* x, int leng) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use conditional branching outside memory access
    if(i < leng) {
        printf("%f,", x[i]);
    }
}