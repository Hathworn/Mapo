#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float logit1(const float x) {
    return expf(x) / (1.0f + expf(x));
}

__global__ void logit(float* y, const float* x, int leng) {
    // Calculate global index for each thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a stride loop to handle out-of-bounds access
    int stride = blockDim.x * gridDim.x;
    for (; i < leng; i += stride) {
        y[i] = logit1(x[i]);
    }
}