#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_long(long* x, int leng) {
    // Use built-in variable for efficiency
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < leng) {
        printf("%ld,", x[i]);
    }
}