#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void div(float *a, float *b, float *c)
{
    // Optimize by using thread idx for parallelism
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    c[idx] = a[idx] / b[idx];
}