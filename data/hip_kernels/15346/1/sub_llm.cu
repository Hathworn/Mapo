#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sub(int *a, int *b, int *c)
{
    // Use threadIdx.x to support multiple blocks
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate subtraction for the entire array
    c[idx] = a[idx] - b[idx];
}