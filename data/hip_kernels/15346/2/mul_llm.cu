#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void mul(int *a, int *b, int *c)
{
    // Use block and thread indices for parallel computation
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform bounds checking
    if (index == 0) 
    {
        *c = *a * *b;
    }
}