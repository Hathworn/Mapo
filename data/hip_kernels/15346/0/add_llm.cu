#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
    // Using shared memory for optimization
    __shared__ int sharedA, sharedB;
    
    if (threadIdx.x == 0) {
        sharedA = *a;
        sharedB = *b;
    }
    __syncthreads();
    
    if (threadIdx.x == 0) {
        *c = sharedA + sharedB;
    }
}