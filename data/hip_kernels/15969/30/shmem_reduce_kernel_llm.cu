#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shmem_reduce_kernel(float * d_out, const float * d_in) {
    extern __shared__ float sdata[];
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // Load input into shared memory
    if (myId < gridDim.x * blockDim.x) {
        sdata[tid] = d_in[myId];
    } else {
        sdata[tid] = 0.0f; // Handle out-of-bound threads
    }
    __syncthreads(); 

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && myId + s < gridDim.x * blockDim.x) { // Bound checking
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); 
    }

    // Write result from the first thread of the block
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}