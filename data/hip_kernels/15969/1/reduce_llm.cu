#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int *s) {
    // Store the current value to shared memory
    int t = threadIdx.x;
    s[t] = p;
    __syncthreads();

    // Perform warp-level reduction
    if (t < 16) s[t] += s[t + 16];
    __syncthreads();
    if (t < 8) s[t] += s[t + 8];
    __syncthreads();
    if (t < 4) s[t] += s[t + 4];
    __syncthreads();
    if (t < 2) s[t] += s[t + 2];
    __syncthreads();
    if (t < 1) s[t] += s[t + 1];
    __syncthreads();

    // Return the final reduction result
    return s[0];
}

__global__ void reduce(unsigned int *d_out_shared, const unsigned int *d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    if (t == 0) {
        *d_out_shared = sr;
    }
}