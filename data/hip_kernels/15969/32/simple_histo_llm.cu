#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Calculate the global index
    int myId = threadIdx.x + blockDim.x * blockIdx.x;

    // Load into register
    int myItem = d_in[myId]; 

    // Calculate bin index
    int myBin = myItem % BIN_COUNT;

    // Atomic increment for bin count
    atomicAdd(&d_bins[myBin], 1);
}