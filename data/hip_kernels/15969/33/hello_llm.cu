#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use a single printf per block for optimized performance
    __shared__ char msg[256];
    int tid = threadIdx.x;
    
    if (tid == 0) {
        printf("Hello world! This is block %d\n", blockIdx.x);
    }
    __syncthreads(); // Ensure all threads reach this point before proceeding
}