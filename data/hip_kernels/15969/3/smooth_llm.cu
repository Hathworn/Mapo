#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth(float * v_new, const float * v) {
    // Use block-level looping to ensure all indices are covered
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < blockDim.x * gridDim.x; i += blockDim.x * gridDim.x) {
        int myLeftIdx = (i == 0) ? 0 : i - 1; // Boundary check for left index
        int myRightIdx = (i == (blockDim.x * gridDim.x - 1)) ? blockDim.x * gridDim.x - 1 : i + 1; // Boundary check for right index
        float myElt = v[i];
        float myLeftElt = v[myLeftIdx];
        float myRightElt = v[myRightIdx];
        v_new[i] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt; // Update kernel logic
    }
}