#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory to reduce the number of printf calls
__global__ void hello()
{
    // Load block ID into shared memory
    __shared__ int shared_block_id;
    if (threadIdx.x == 0)
    {
        shared_block_id = blockIdx.x;
    }
    __syncthreads();
    
    // Only print from one thread per block to reduce console output overhead
    if (threadIdx.x == 0)
    {
        printf("Hello world! I'm a thread in block %d\n", shared_block_id);
    }
}