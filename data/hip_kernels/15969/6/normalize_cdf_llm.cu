#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
{
    const float normalization_constant = 1.f / d_input_cdf[n - 1];
    int global_index_1d = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Optimize memory access by checking the boundary first
    if (global_index_1d >= n) return; 

    // Efficiently load and process values
    unsigned int input_value = d_input_cdf[global_index_1d];
    float output_value = __fmul_rn(input_value, normalization_constant);  // Use fast multiplication

    d_output_cdf[global_index_1d] = output_value;
}