#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void naive_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Shared memory for storing histogram for each block
    extern __shared__ int shared_bins[];
    int thread_id = threadIdx.x;
    int block_start_index = thread_id + blockDim.x * blockIdx.x;

    // Initialize shared histogram
    if (thread_id < BIN_COUNT)
    {
        shared_bins[thread_id] = 0;
    }
    __syncthreads();

    // Calculate histogram index and update shared histogram
    int item = d_in[block_start_index];
    int bin = item % BIN_COUNT;

    atomicAdd(&shared_bins[bin], 1);
    __syncthreads();

    // Merge shared histogram into global histogram
    if (thread_id < BIN_COUNT)
    {
        atomicAdd(&d_bins[thread_id], shared_bins[thread_id]);
    }
}