#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cubic(float* d_out, float* d_in)
{
    // Use blockIdx and blockDim for larger input arrays
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check within bounds before accessing memory
    if (idx < N) 
    {
        float f = d_in[idx];
        d_out[idx] = f * f * f; // Compute cube more efficiently
    }
}