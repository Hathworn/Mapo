#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
    extern __shared__ float s_data[]; // Allocate shared memory dynamically

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // Move data from global memory to shared memory
    s_data[tid] = d_in[myId];
    __syncthreads(); // Ensure all threads have loaded their data

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            s_data[tid] += s_data[tid + s]; // Reduce using shared memory
        }
        __syncthreads(); // Synchronize for next stage
    }

    // Only thread 0 writes the result for this block back to global memory
    if (tid == 0)
    {
        d_out[blockIdx.x] = s_data[0];
    }
}