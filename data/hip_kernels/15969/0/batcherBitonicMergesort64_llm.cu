#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    for (int stage = 0; stage <= 5; stage++)
    {
        // Calculate the distance for the current stage
        int distance = 1 << stage;
        for (int substage = stage; substage >= 0; substage--)
        {
            // Calculate the bitonic merge index
            int index = 1 << substage;

            // Calculate the condition for swapping
            int pair_index = tid ^ index;
            if (substage == stage || (pair_index > tid && pair_index < 64))
            {
                // Perform comparison and swapping
                if ((tid & distance) == 0)
                {
                    if (sdata[tid] > sdata[pair_index])
                    {
                        float temp = sdata[tid];
                        sdata[tid] = sdata[pair_index];
                        sdata[pair_index] = temp;
                    }
                }
                else
                {
                    if (sdata[tid] < sdata[pair_index])
                    {
                        float temp = sdata[tid];
                        sdata[tid] = sdata[pair_index];
                        sdata[pair_index] = temp;
                    }
                }
            }
            __syncthreads(); // Ensure all threads complete swap before next iteration
        }
    }

    d_out[tid] = sdata[tid];
}