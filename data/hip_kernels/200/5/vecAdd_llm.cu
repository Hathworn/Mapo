#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    // Efficiently calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within array bounds
    if (i < len) {
        // Perform vector addition
        out[i] = in1[i] + in2[i];
    }
}