#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 100000000

__global__ void daxpy_simple(int n, double alpha, double *x, double *y) {
    // Use a loop to process multiple elements per thread
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < n; idx += blockDim.x * gridDim.x) {
        y[idx] += alpha * x[idx];
    }
}