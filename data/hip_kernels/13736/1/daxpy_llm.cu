#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 100000000

__global__ void daxpy(int n, double alpha, double *x, double *y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Use a stride loop to process elements more efficiently
    for (; idx < n; idx += stride) {
        y[idx] += alpha * x[idx];
    }
}