#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// wrapper for a call option
__global__ void mc_kernel_call(float * d_s, float T, float K, float S0, float sigma, float mu, float r, float dt, float * d_normals, unsigned N_STEPS, unsigned N_PATHS)
{
    // Cache thread and block information
    const unsigned tid = threadIdx.x;
    const unsigned bid = blockIdx.x;
    const unsigned bsz = blockDim.x;

    // Compute global index
    int global_idx = tid + bid * bsz;
    
    // Early return if index is out of bounds
    if (global_idx >= N_PATHS) return;

    // Initialize variables
    float s_curr = S0;
    int n_idx = global_idx;

    // Simulation loop
    for (unsigned n = 0; n < N_STEPS; ++n) {
        s_curr += mu * s_curr * dt + sigma * s_curr * d_normals[n_idx];
        n_idx += bsz * gridDim.x; // Advance to the next step's normal value
    }

    // Calculate payoff
    float payoff = (s_curr > K) ? (s_curr - K) : 0.0f;
    
    // Store the discounted payoff
    d_s[global_idx] = expf(-r * T) * payoff;
}