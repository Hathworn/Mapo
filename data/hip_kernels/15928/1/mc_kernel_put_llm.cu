#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mc_kernel_put(float* d_s, float T, float K, float S0, float sigma, float mu, float r, float dt, float* d_normals, unsigned N_STEPS, unsigned N_PATHS)
{
    const unsigned tid = threadIdx.x;
    const unsigned bid = blockIdx.x;
    const unsigned bsz = blockDim.x;
    
    int s_idx = tid + bid * bsz;
    int n_idx = s_idx; // Use same index for simplicity
    float s_curr = S0;
    double payoff;

    if (s_idx < N_PATHS) {
        // Unroll the loop for better performance
        #pragma unroll
        for (int n = 0; n < N_STEPS; ++n) {
            s_curr += mu * s_curr * dt + sigma * s_curr * d_normals[n_idx];
            n_idx++;
        }
        payoff = (s_curr < K ? K - s_curr : 0.0);
        d_s[s_idx] = exp(-r * T) * payoff; // Move result computation outside if statement
    }
}