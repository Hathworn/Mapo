#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UpdateScalars(float *WHAT, float *WITH, float AMOUNT, float *MASS) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < blockDim.x * gridDim.x) { // Bounds check for thread idx
        float mass_value = MASS[idx];   // Cache MASS value to register
        if (mass_value != 0.0f) {       // Check to avoid division by zero
            WHAT[idx] += AMOUNT * WITH[idx] / mass_value;
        }
    }
}