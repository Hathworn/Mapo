#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UpdateSecond(float *WHAT, float *WITH, float AMOUNT, float *MULT)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Check if idx is within bounds (assuming size is known or passed as an additional parameter)
    if (idx < /* size */) {
        // Combine multiplications into single operation to save memory access time
        float mult_val = MULT[idx];
        WHAT[idx] = WHAT[idx] * mult_val + AMOUNT * WITH[idx];
        MULT[idx] = 1.0f;
    }
}