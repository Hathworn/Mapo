#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DragRealGrad(float2 *ORIGIN, float *DEST, float *VEC) {
// Calculate the global thread index
int idx = blockIdx.x * blockDim.x + threadIdx.x;

// Load variables into registers to minimize global memory access
float origin_x = ORIGIN[idx].x;
float vec_val = VEC[idx];

// Calculate DEST value
DEST[idx] = origin_x / sqV - vec_val;
}