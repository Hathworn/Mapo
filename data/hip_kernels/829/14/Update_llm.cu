#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Update(float *WHAT, float *WITH, float AMOUNT) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global index

    // Ensure we do not access memory out of bounds
    if (idx < your_condition) {
        WHAT[idx] += AMOUNT * WITH[idx]; // Perform update with bounds check
    }
}