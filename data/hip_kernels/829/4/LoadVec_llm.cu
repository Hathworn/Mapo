#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LoadVec(float *vector, float2 *FFT) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) { // Boundary check to prevent out-of-range access
        float val = vector[idx]; // Load vector element once
        FFT[idx] = make_float2(val, 0.0f); // Use make_float2 for cleaner assignment
    }
}