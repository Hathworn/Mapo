#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Print(float *beta, float *sigma, float *rho, int iter )
{
    // Loop unrolling for reduced instruction overhead.
    for (int i = 0; i < 3; ++i) {
        printf("\n %d -- %d) b %.5f -- s %.5f -- r %.5f ", iter, i + 1, beta[i], sigma[i], rho[i]);
    }
}