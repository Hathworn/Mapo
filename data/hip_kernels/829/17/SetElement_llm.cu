#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel using block and thread indexing
__global__ void SetElement(float *vector, int position, float what) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == position) {
        vector[idx] = what;
    }
}