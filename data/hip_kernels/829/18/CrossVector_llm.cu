#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CrossVector(float *first , float *second) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < N) { // Ensure index is within bounds
        // Use fast intrinsic sqrt function for better performance
        first[idx] *= __fsqrt_rn(second[idx]);
    }
}