#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LoadAddVecSecond(float *vector, float2 *FFT) {
    // Calculate global index for each thread in the grid
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 

    // Load vector value once for better memory access efficiency
    float vecVal = vector[idx] / sqV;

    // Perform scaling on FFT components
    FFT[idx].x *= vecVal;
    FFT[idx].y *= vecVal;
}