#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Replace(const float * __restrict__ WHAT, float * __restrict__ WHERE)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < gridDim.x * blockDim.x) // Check bounds
    {
        WHERE[idx] = WHAT[idx]; // Copy data from WHAT to WHERE
    }
}