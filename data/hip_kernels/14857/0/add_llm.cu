#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 256

__global__ void add(int *a, int *b, int *c, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Use both blockIdx and threadIdx for proper indexing
    if (tid < N) {
        c[tid] = a[tid] + b[tid]; // Add vectors element-wise
    }
}