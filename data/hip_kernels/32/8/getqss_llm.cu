#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getqss(double *IN, double *qss, int N, int t) {
    // Use direct index calculation and avoid unnecessary variables
    int I = 10 * N + 10; 
    qss[t] = IN[I];
}