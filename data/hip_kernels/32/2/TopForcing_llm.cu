#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TopForcing(double ppt, double *eff_rain, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;  // Calculate stride once
    for (int i = tid; i < size; i += stride) {  // Use for loop for better readability
        eff_rain[i] = ppt;  // Assign ppt to eff_rain directly
    }
}