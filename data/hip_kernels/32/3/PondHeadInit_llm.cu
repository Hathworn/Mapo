#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void PondHeadInit(double *ph, int size) {
    // Calculate unique global thread id
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;

    // Use loop unrolling for better performance
    for (int i = tid; i < size; i += gridSize) {
        ph[i] = psi_min;
    }
}