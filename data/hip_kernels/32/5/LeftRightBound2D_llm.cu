#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LeftRightBound2D(double *Hs, double *Ztopo, double *K2e, double *K2w, int BC2D, int M, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;  // Precompute the stride to optimize access
    while (tid < M) {
        int tidN = tid * N;  // Precompute to optimize repeated memory access
        // no-flow BCs
        if (BC2D == 0) {
            Hs[tidN] = Hs[tidN + 1];
            Hs[tidN + N - 1] = Hs[tidN + N - 2];
        } else {  // Critical depth flow BCs
            Hs[tidN] = hcri + Ztopo[tidN];
            Hs[tidN + N - 1] = hcri + Ztopo[tidN + N - 1];
        }
        
        K2w[tidN] = K2w[tidN + 1];
        K2e[tidN + N - 1] = K2e[tidN + N - 2];
        tid += stride;  // Use precomputed stride
    }
}