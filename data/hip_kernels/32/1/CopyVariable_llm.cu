#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CopyVariable(double *var_in, double *var_out, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Pre-compute stride for optimization
    for (int i = tid; i < size; i += stride) { // Use for-loop instead of while for potential unrolling
        var_out[i] = var_in[i];
    }
}