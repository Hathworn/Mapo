#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VarPrint(double *Var, int M, int N, int P) {
    int k = blockIdx.z * blockDim.z + threadIdx.z; // Optimize: Use 3D grid for P
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Optimize: Use 3D grid for N
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Optimize: Use 3D grid for M
    
    if (k < P && i < N && j < M) {
        printf("%4.3f ", Var[k * M * N + i * M + j]);
        if (j == M - 1) 
            printf("\n"); // Ensure newline after each row
    }
    if (k < P && i == N - 1 && j == 0)
        printf("\n\n"); // Ensure double newline after each matrix
}