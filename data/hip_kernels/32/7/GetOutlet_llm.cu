#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GetOutlet(double *h, double *houtlet, double *u, double *uout, double *v, double *vout, int M, int N, int t) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride outside loop

    int offset = N - 2; // Precompute offset as it doesn't change inside loop
    while (tid < M) {
        int index = (tid + 1) * N - offset; // Compute index only once
        houtlet[t * M + tid] = h[index];
        vout[t * M + tid] = v[index];
        uout[t * M + tid] = u[index];
        tid += stride;
    }
}