#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = gridDim.x * blockDim.x;
    // Loop to enhance data processing for larger arrays
    for (; i < N; i += totalThreads) {
        Y[i * INCY] *= X[i * INCX];
    }
}