#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID
    if(tid >= N) return;

    int in_s = tid % spatial;
    int remaining = tid / spatial;
    int in_c = remaining % layers;
    int b = remaining / layers;

    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;

    if (forward) {
        out[i2] = x[i1];
    } else {
        out[i1] = x[i2];
    }
}