#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;
    
    // Unroll the loops for better memory access patterns
    for(int b = 0; b < batch; ++b){
        for(int i = p; i < size; i += BLOCK){
            int index = i + size * (filter + n * b);
            sum += delta[index] * x_norm[index];
        }
    }
    part[p] = sum;
    __syncthreads();
    
    // Use a reduction pattern for better efficiency
    if (p == 0) {
        float block_sum = 0;
        for(int i = 0; i < BLOCK; ++i) {
            block_sum += part[i];
        }
        atomicAdd(&scale_updates[filter], block_sum);
    }
}