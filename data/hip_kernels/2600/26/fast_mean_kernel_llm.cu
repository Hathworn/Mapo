#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0.0f;  // Initialize shared memory

    int filter = blockIdx.x;
    int numElements = batch * spatial;

    // Unroll loop for performance
    for (int j = 0; j < batch; ++j) {
        int baseIndex = j * spatial * filters + filter * spatial + id;
        for (int i = 0; i < spatial; i += threads) {
            int index = baseIndex + i;
            if (index < numElements) {
                local[id] += x[index];  // Accumulate values
            }
        }
    }

    // Use atomic addition for reduction to prevent race condition
    if (id == 0) {
        float sum = 0.0f;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        mean[filter] = sum / (spatial * batch);
    }
}