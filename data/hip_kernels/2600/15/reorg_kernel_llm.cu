#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if(i >= N) return;

    // Compute input coordinates
    int in_index = i;
    int in_w = i % w;
    i /= w;
    int in_h = i % h;
    i /= h;
    int in_c = i % c;
    i /= c;
    int b = i % batch;

    // Calculate output channel and offset
    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;

    // Compute output coordinates
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    // Assign values based on direction
    if(forward) {
        out[out_index] = x[in_index];
    } else {
        out[in_index] = x[out_index];
    }
}