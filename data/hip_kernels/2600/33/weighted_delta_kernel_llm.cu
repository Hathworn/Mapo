#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Process only if index is within bounds
    if (i < n) {
        float dc_i = dc[i]; // Load dc[i] once
        float s_i = s[i];   // Load s[i] once
        
        if (da) {
            da[i] += dc_i * s_i; // Update da if not null
        }
        
        // Update db and ds
        db[i] += dc_i * (1 - s_i);
        ds[i] += dc_i * (a[i] - b[i]);
    }
}