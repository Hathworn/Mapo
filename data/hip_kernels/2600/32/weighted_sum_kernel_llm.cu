#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D grid for simplicity
    int totalThreads = blockDim.x * gridDim.x; // Calculate total number of threads
    for (; i < n; i += totalThreads) { // Loop to cover all elements
        float b_val = (b != nullptr) ? b[i] : 0; // Conditional b element retrieval
        c[i] = s[i] * a[i] + (1.0f - s[i]) * b_val; // Calculate weighted sum
    }
}