#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = gridDim.x * blockDim.x; // Calculate total number of threads
    
    while (id < size) {
        int i = id % minw;
        int j = (id / minw) % minh;
        int k = (id / (minw * minh)) % minc;
        int b = id / (minw * minh * minc);

        int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
        int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));
        out[out_index] += add[add_index];
        
        id += gridSize; // Move to next element in increments of total threads
    }
}