#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (i >= filters) return;

    float sum = 0.0f; // Use a local variable for accumulation
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += delta[index];
        }
    }
    mean_delta[i] = sum * (-1.0f / sqrtf(variance[i] + 1e-6f)); // Use sqrtf for float and simplify constant
}