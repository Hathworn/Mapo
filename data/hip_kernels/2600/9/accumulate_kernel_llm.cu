#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    // Use a temporary variable for accumulation to optimize memory access
    float temp_sum = 0.0f;
    for (int k = 0; k < n; ++k) {
        temp_sum += x[k * groups + i];
    }

    // Store the result in global memory
    sum[i] = temp_sum;
}