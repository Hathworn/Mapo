#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Ensure kernel operation is performed only for valid threads
    if (i * INCX < N) {
        X[i * INCX] = ALPHA;
    }
}