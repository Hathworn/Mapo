#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    __shared__ float local[BLOCK];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0;

    // Loop over batches and spatial parts, reducing memory accesses with larger steps
    for(int j = 0; j < batch; ++j) {
        for(int i = id; i < spatial; i += BLOCK) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += delta[index];
        }
    }

    // Sum local results using parallel reduction in shared memory
    __syncthreads();
    for (int stride = BLOCK / 2; stride > 0; stride /= 2) {
        if (id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    // Write result to global memory
    if (id == 0) {
        mean_delta[filter] = local[0] * (-1./sqrt(variance[filter] + .000001f));
    }
}