#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Compute indices more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float local_mean = 0.0f;
    float scale = 1.0f / (batch * spatial);

    // Combine loops to improve data locality
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            // Calculate index within the flattened array
            int index = j * filters * spatial + i * spatial + k;
            local_mean += x[index];
        }
    }

    // Scale the accumulated sum
    mean[i] = local_mean * scale;
}