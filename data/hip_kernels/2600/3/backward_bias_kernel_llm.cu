#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0.0f;

    // Loop over batch and size with stride BLOCK
    for(int b = 0; b < batch; ++b){
        for(int i = p; i < size; i += BLOCK){
            int index = i + size * (filter + n * b);
            sum += delta[index];
        }
    }
    part[p] = sum;

    // Reduce within block
    __syncthreads();
    if (BLOCK >= 1024) { if (p < 512) part[p] += part[p + 512]; __syncthreads(); }
    if (BLOCK >= 512) { if (p < 256) part[p] += part[p + 256]; __syncthreads(); }
    if (BLOCK >= 256) { if (p < 128) part[p] += part[p + 128]; __syncthreads(); }
    if (BLOCK >= 128) { if (p < 64) part[p] += part[p + 64]; __syncthreads(); }
    
    if (p < 32) {
        // Unrolling warp here
        if (BLOCK >= 64) part[p] += part[p + 32];
        if (BLOCK >= 32) part[p] += part[p + 16];
        if (BLOCK >= 16) part[p] += part[p + 8];
        if (BLOCK >= 8) part[p] += part[p + 4];
        if (BLOCK >= 4) part[p] += part[p + 2];
        if (BLOCK >= 2) part[p] += part[p + 1];
    }

    // Update bias_updates in the last thread of the warp
    if (p == 0) {
        bias_updates[filter] += part[0];
    }
}