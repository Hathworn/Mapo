#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;

    // Use warp-level reduction to find the largest value
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        float val = input[i];
        atomicMax(&largest, val);
    }
    __syncthreads();

    // Calculate exponentials and sum them
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        float e = exp(input[i] / temp - largest / temp);
        atomicAdd(&sum, e);
        output[i] = e;
    }
    __syncthreads();

    // Normalize the result
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        output[i] /= sum;
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output)
{
    int b = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (b >= batch) return;
    softmax_device(n, input + b * offset, temp, output + b * offset);
}