#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[BLOCK];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    // Initialize local sum to 0
    local[id] = 0.0f;

    int index_base = filter * spatial;

    // Use striding for performance
    for(int j = 0; j < batch; ++j){
        for(int i = id; i < spatial; i += threads){
            int index = j * spatial * filters + index_base + i;
            local[id] += delta[index] * (x[index] - mean[filter]);
        }
    }

    __syncthreads();

    // Reduction within block
    if(id < 32){
        for(int offset = 32; offset < threads; offset += 32){
            local[id] += local[id + offset];
        }
    }

    __syncthreads();

    // Final reduction and update of variance_delta
    if(id == 0){
        float sum = 0.0f;
        for(int i = 0; i < 32; ++i){
            sum += local[i];
        }
        variance_delta[filter] = sum * -0.5f * powf(variance[filter] + 0.000001f, -1.5f);
    }
}