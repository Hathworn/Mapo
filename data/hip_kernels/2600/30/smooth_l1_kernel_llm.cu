#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global thread index
    if(i < n){
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff); // Use fast math function for absolute value
        if(abs_val < 1.0f) {
            error[i] = diff * diff;
            delta[i] = diff;
        }
        else {
            error[i] = 2.0f * abs_val - 1.0f;
            delta[i] = copysignf(1.0f, diff); // Use copysignf for sign function
        }
    }
}