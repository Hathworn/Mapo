#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate linear index
    int gridSize = gridDim.x * blockDim.x;          // Calculate total grid size

    while (i < N) {                                 // Loop over entire range
        X[i * INCX] *= ALPHA;                       // Scale element
        i += gridSize;                              // Move to next element in the grid
    }
}