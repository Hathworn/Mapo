#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    // Calculate unique global index for each thread
    int index = blockIdx.x * blockDim.x + threadIdx.x; 
    if (index >= N) return;

    // Precompute constant terms outside of the per-element operations
    float correction1 = 1.0f - powf(B1, t);
    float correction2 = sqrtf(1.0f - powf(B2, t));
    
    // Optimal update calculation for x[i]
    float m_hat = m[index] / correction1;
    float v_hat = sqrtf(v[index]) / correction2;

    x[index] = x[index] - (rate * m_hat / (v_hat + eps));
}