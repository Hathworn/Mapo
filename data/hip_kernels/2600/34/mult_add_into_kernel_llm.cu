#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = gridDim.x * blockDim.x;
    
    // Loop with grid stride to handle large data sets efficiently
    for(; i < n; i += gridStride) {
        c[i] += a[i] * b[i];
    }
}