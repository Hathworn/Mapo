#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;
    
    float sum = 0.0f, norm1 = 0.0f, norm2 = 0.0f;

    // Use register variables to reduce memory access latency
    float output_val1, output_val2;

    // Loop unrolling for batch size to improve performance
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            output_val1 = output[i1];
            output_val2 = output[i2];
            sum += output_val1 * output_val2;
            norm1 += output_val1 * output_val1;
            norm2 += output_val2 * output_val2;
        }
    }
    norm1 = sqrtf(norm1);
    norm2 = sqrtf(norm2);
    float norm = norm1 * norm2;
    sum = sum / norm;

    // Reuse calculated values and avoid redundant calculations
    float scale_sum = -scale * sum / norm;

    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            delta[i1] += scale_sum * output[i2];
            delta[i2] += scale_sum * output[i1];
        }
    }
}