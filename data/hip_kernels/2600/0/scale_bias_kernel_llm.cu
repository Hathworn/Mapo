#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int filter = blockIdx.y;
    int batch = blockIdx.z;
    int baseIdx = (batch * n + filter) * size; // Compute base index once to optimize

    for (int offset = blockIdx.x * blockDim.x + threadIdx.x; offset < size; offset += blockDim.x * gridDim.x) {
        // Use a loop to handle more work per thread and improve memory access efficiency
        output[baseIdx + offset] *= biases[filter];
    }
}