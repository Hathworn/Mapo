#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Combine index computation for clarity
    int stride = gridDim.x * blockDim.x;            // Compute stride value
    for(int idx = i; idx < N; idx += stride) {      // Loop through all elements with stride
        X[idx * INCX] = ALPHA;                      // Assign ALPHA to each element
    }
}