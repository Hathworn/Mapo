#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate unique global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // stride loop to process entire array
    for(int i = idx; i < N; i += blockDim.x * gridDim.x) {
        // Calculate index with stride
        int index = i * INCX;
        
        // Check condition and update element
        if((X[index] * X[index]) < (ALPHA * ALPHA)) {
            X[index] = 0;
        }
    }
}