#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    int stride = gridDim.x * blockDim.x; // Stride for grid-stride loop

    for (int i = idx; i < N; i += stride) {
        Y[i * INCY] = powf(X[i * INCX], ALPHA); // Use powf for single precision
    }
}