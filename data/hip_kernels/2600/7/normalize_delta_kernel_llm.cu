#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop to handle larger data uses stride
    for (; index < N; index += stride) {
        int f = (index / spatial) % filters;
        float var_sqrt = sqrt(variance[f]) + .000001f; // Precompute square root of variance
        float common_term = 1.0f / (spatial * batch); // Precompute common division
        delta[index] = delta[index] * 1.0f / var_sqrt 
                     + variance_delta[f] * 2.0f * (x[index] - mean[f]) * common_term 
                     + mean_delta[f] * common_term;
    }
}