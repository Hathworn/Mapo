#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    // Calculate the global thread index
    int offset = blockDim.x * (blockIdx.z * gridDim.y + blockIdx.y) + threadIdx.x;

    if (offset < size) {
        // Calculate index with fused batch and filter computation
        int index = (offset / size) * size * n + (offset % size);
        output[index] += biases[offset / size];
    }
}