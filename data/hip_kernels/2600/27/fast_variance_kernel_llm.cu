#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0.0f;

    int filter = blockIdx.x;
    int sharedIdx = threadIdx.x;
    int blockStride = blockDim.x;

    for (int j = 0; j < batch; ++j) {
        for (int i = sharedIdx; i < spatial; i += blockStride) { // Use strided iteration
            int index = j * spatial * filters + filter * spatial + i;
            if (i < spatial) {
                local[id] += powf((x[index] - mean[filter]), 2); // Access each element within bounds
            }
        }
    }

    __syncthreads(); // Ensure all partial sums are complete

    if (id == 0) {
        variance[filter] = 0;
        for (int i = 0; i < threads; ++i) {
            variance[filter] += local[i];
        }
        variance[filter] /= (spatial * batch - 1);
    }
}