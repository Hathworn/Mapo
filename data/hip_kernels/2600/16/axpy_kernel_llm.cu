#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Stride through elements with grid dimension for larger data processing
    for (; i < N; i += blockDim.x * gridDim.x) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}