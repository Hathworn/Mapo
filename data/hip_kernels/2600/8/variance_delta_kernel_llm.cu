#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Calculate the global index for the filter
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float var_val = variance[i];
    float mean_val = mean[i];
    float var_delta = 0.0f;

    // Combine the batch and spatial loops for increased thread utilization
    for (int j = 0; j < batch * spatial; ++j) {
        int index = j * filters + i;
        var_delta += delta[index] * (x[index] - mean_val);
    }
    
    // Calculate the variance delta for the filter
    variance_delta[i] = var_delta * -.5f * pow(var_val + .000001f, -1.5f);
}