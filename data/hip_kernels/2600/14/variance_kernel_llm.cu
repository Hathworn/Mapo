#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Calculate scale factor once
    float scale = 1.0f / (batch * spatial - 1);
    // Calculate global thread ID
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    
    if (i >= filters) return;
    
    // Initialize intermediate variable for variance
    float var = 0.0f;
    
    // Use shared memory if possible to reduce global memory access
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            // Accumulate variance on local variable
            float diff = x[index] - mean[i];
            var += diff * diff;
        }
    }
    
    // Write the scaled variance value to the global output array
    variance[i] = var * scale;
}