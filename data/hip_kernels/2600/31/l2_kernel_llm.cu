#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Using blockIdx.x and distributing grid properly for better performance
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float diff = truth[i] - pred[i];
        error[i] = diff * diff;
        delta[i] = diff;
    }
}