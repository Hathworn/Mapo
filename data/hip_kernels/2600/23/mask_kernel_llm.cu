#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}


__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    // Use shared memory to reduce global memory access
    __shared__ float shared_mask;
    if (threadIdx.x == 0) {
        shared_mask = mask_num;
    }
    __syncthreads();

    // Check bounds and perform operation
    if (i < n && mask[i] == shared_mask) {
        x[i] = shared_mask;
    }
}