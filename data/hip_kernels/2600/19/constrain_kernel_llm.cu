#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate a unique index for each thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within the bounds
    if(i < N) {
        float val = X[i * INCX];  // Load the value once
        // Apply the constraint and store the result
        X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, val));
    }
}