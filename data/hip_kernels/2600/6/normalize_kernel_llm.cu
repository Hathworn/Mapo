#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

// Optimized kernel function
__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for larger data sets

    for (int i = index; i < N; i += stride) { // Use loop unrolling for better memory access
        int f = (i / spatial) % filters;
        x[i] = (x[i] - mean[f]) / (sqrt(variance[f]) + .000001f);
    }
}