#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Use shared memory to improve memory access efficiency
    __shared__ float sharedA[1024];  // Assumes blockDim.x <= 1024
    __shared__ float sharedB[1024];  // Assumes blockDim.x <= 1024

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        // Load data into shared memory
        sharedA[threadIdx.x] = a[i];
        sharedB[threadIdx.x] = b[i];
        __syncthreads();  // Ensure all threads have loaded their data

        // Perform addition
        dest[i] = sharedA[threadIdx.x] + sharedB[threadIdx.x];
    }
}