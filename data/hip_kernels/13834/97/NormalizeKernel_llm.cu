#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Use shared memory to cache normalization factor
    __shared__ float shared_normalization_factor[BLOCK_SIZE_X * BLOCK_SIZE_Y];
    int threadIndex = threadIdx.y * blockDim.x + threadIdx.x;
  
    if (i < h && j < w) {
        const int pos = i * s + j;
        shared_normalization_factor[threadIndex] = normalization_factor[pos];
        __syncthreads();

        // Optimize for read access from shared memory
        float scale = shared_normalization_factor[threadIndex];
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);
        image[pos] *= invScale;
    }
}
```
