#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst) {
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate target pixel position
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // Decompose pixel position into integer and decimal parts
    int tx, ty;
    float dx = modff(cx, reinterpret_cast<float*>(&tx));
    float dy = modff(cy, reinterpret_cast<float*>(&ty));

    // Ensure tx, ty are within bounds
    if (tx < 0 || tx >= w || ty < 0 || ty >= h) return;

    float value = src[image_row_offset + j];

    // Use shared memory for partial pixel contributions to reduce global memory operations
    __shared__ float shared_dst[blockDim.y][blockDim.x];
    __shared__ float shared_norm[blockDim.y][blockDim.x];

    // Initialize shared memory
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        shared_dst[threadIdx.y][threadIdx.x] = 0.0f;
        shared_norm[threadIdx.y][threadIdx.x] = 0.0f;
    }
    __syncthreads();

    // Compute and accumulate weights and values for each corner
    float weight;
    weight = dx * dy;
    atomicAdd(&shared_dst[threadIdx.y][threadIdx.x], value * weight);
    atomicAdd(&shared_norm[threadIdx.y][threadIdx.x], weight);

    tx -= 1;
    weight = (1.0f - dx) * dy;
    if (tx >= 0) {
        atomicAdd(&shared_dst[threadIdx.y][threadIdx.x], value * weight);
        atomicAdd(&shared_norm[threadIdx.y][threadIdx.x], weight);
    }

    ty -= 1;
    weight = (1.0f - dx) * (1.0f - dy);
    if (tx >= 0) {
        atomicAdd(&shared_dst[threadIdx.y][threadIdx.x], value * weight);
        atomicAdd(&shared_norm[threadIdx.y][threadIdx.x], weight);
    }

    tx += 1;
    weight = dx * (1.0f - dy);
    if (tx >= 0) {
        atomicAdd(&shared_dst[threadIdx.y][threadIdx.x], value * weight);
        atomicAdd(&shared_norm[threadIdx.y][threadIdx.x], weight);
    }

    // Synchronize and writeback updates to global memory
    __syncthreads();
    
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        atomicAdd(&dst[ty * image_stride + tx], shared_dst[threadIdx.y][threadIdx.x]);
        atomicAdd(&normalization_factor[ty * image_stride + tx], shared_norm[threadIdx.y][threadIdx.x]);
    }
}