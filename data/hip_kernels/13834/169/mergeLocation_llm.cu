#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single conditional check to reduce divergence.
    if (ptidx >= npoints) return;
    
    // Load data into registers for better memory access efficiency.
    short2 loc = loc_[ptidx];

    // Perform computation using registers.
    x[ptidx] = loc.x * scale;
    y[ptidx] = loc.y * scale;
}