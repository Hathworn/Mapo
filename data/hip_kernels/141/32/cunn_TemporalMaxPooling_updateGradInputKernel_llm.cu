#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate offsets
    int timeIndex = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    if (timeIndex >= output_w) return; // Early exit for out-of-bounds threads

    int batchOffset = blockIdx.x * input_w * input_n;
    int gradInputOffset = timeIndex * input_n * dW;
    int gradOutputOffset = timeIndex * input_n;
    int indicesOffset = timeIndex * input_n;

    float *gradInput_data = gradInput + batchOffset + gradInputOffset;
    float *gradOutput_data = gradOutput + batchOffset + gradOutputOffset;
    float *indices_data = indices + batchOffset + indicesOffset;

    // Update gradInput for each feature
    for (int feat = 0; feat < input_n; ++feat) {
        int index = static_cast<int>(indices_data[feat]) * input_n + feat;
        atomicAdd(&gradInput_data[index], gradOutput_data[feat]); // Use atomicAdd to avoid race conditions
    }
}