#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void min_gradInput(float *input, float *output, float *indices, long nrows, long ncols)
{
    // Calculate output offset using ID comparison with total nrows instead of extra condition
    long o = blockIdx.x * blockDim.x + threadIdx.x;
    if (o < nrows)
    {
        // Calculate input offset once
        long i = o * ncols;
        
        // Calculate index directly and assign
        long idx = __float2int_rd(indices[o]) - 1;
        input[i + idx] = output[o];
    }
}