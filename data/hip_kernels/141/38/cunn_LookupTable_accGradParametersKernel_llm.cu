#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_LookupTable_accGradParametersKernel(float *input, float *indices, float *gradOutput, float *gradWeight, float *count, float defaultScale, long numel, long stride) {

    int idx = blockIdx.x * blockDim.y + threadIdx.y; // Modified index calculation for better parallelization

    const int SZ = 4;

    if (idx < numel && (idx == 0 || input[idx] != input[idx - 1])) {
        do {
            const int startFeature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
            const int weightRow = ((int) input[idx] - 1) * stride;
            const int gradOutputRow = ((int) indices[idx] - 1) * stride;
            const float scale = count ? defaultScale / count[idx] : defaultScale;

            float gradient[SZ];
            float weight[SZ];

            #pragma unroll
            for (int ii = 0; ii < SZ; ii++) {
                int featureDim = startFeature + ii * WARP_SIZE; 
                if (featureDim < stride) {
                    gradient[ii] = gradOutput[gradOutputRow + featureDim];
                    weight[ii] = gradWeight[weightRow + featureDim];
                }
            }

            #pragma unroll
            for (int ii = 0; ii < SZ; ii++) {
                weight[ii] += gradient[ii] * scale;
            }

            #pragma unroll
            for (int ii = 0; ii < SZ; ii++) {
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride) {
                    gradWeight[weightRow + featureDim] = weight[ii];
                }
            }

            idx++;
        } while (idx < numel && input[idx] == input[idx - 1]);
    }
}