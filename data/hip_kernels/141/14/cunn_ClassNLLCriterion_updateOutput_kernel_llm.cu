#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_ClassNLLCriterion_updateOutput_kernel(float *output, float *input, float *target, int nframe, int ndim, int sizeAverage, int ntarget) {
    __shared__ float shInputs[NTHREADS];

    int tid = threadIdx.x;
    shInputs[tid] = 0.0f;

    // Unrolling loops can be beneficial here
    for (int i = tid; i < nframe; i += NTHREADS) {
        for (int j = 0; j < ntarget; ++j) {
            int t = (int)target[i * ntarget + j] - 1;
            if (t >= 0) {
                // Fetch input once to reduce global memory access
                float val = input[i * ndim + t];
                shInputs[tid] += val;
            }
        }
    }
    __syncthreads();

    // Reduction step - All threads contribute their partial sums
    if (tid == 0) {
        float sum = 0.0f;
        for (int i = 0; i < NTHREADS; ++i) {
            sum += shInputs[i];
        }
        if (sizeAverage)
            sum /= nframe;
        *output = -sum;
    }
}