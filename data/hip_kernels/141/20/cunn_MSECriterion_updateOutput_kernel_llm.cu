#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define MSECRITERION_THREADS 1024

__global__ void cunn_MSECriterion_updateOutput_kernel(float* output, float *input, float *target, int nframe, int dim, int sizeAverage)
{
    __shared__ float buffer[MSECRITERION_THREADS];
    int k = blockIdx.x;
    float *input_k = input + k * dim;
    float *target_k = target + k * dim;

    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;

    // Parallelized mse calculation with buffer accumulation
    float sum = 0.0f;
    for (int i = i_start; i < i_end; i += i_step)
    {
        float z = input_k[i] - target_k[i];
        sum += z * z;
    }
    buffer[threadIdx.x] = sum;
    __syncthreads();

    // Efficient reduction using shared memory
    if (threadIdx.x == 0)
    {
        float blockSum = 0.0f;
        for (int i = 0; i < blockDim.x; i++)
        {
            blockSum += buffer[i];
        }
        if (sizeAverage)
        {
            blockSum /= dim;
        }
        atomicAdd(output, blockSum); // Use atomicAdd for thread-safe accumulation
    }
}