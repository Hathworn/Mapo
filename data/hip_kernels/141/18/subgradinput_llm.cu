#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinput(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;

    // Get pointers for input/output plane
    gradOutput = gradOutput + o * output_w * output_h;
    gradInput = gradInput + i * input_w * input_h;

    // Get weight
    float the_weight = weight[k];

    // Use shared memory for weights (optimization)
    extern __shared__ float shared_weight[];
    shared_weight[threadIdx.x] = the_weight;
    __syncthreads();

    // Compute gradInput
    for (int yy = yy_start; yy < output_h; yy += blockDim.y * gridDim.y)
    {
        for (int xx = xx_start; xx < output_w; xx += blockDim.x)
        {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput * shared_weight[threadIdx.x];
            
            for (int ky = 0; ky < kH; ky++)
            {
                for (int kx = 0; kx < kW; kx++)
                {
                    ptr_gradInput[kx] += z;
                }
                ptr_gradInput += input_w;
            }
        }
    }
}