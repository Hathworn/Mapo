#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void min_output(float *input, float *output, float *indices, long nrows, long ncols)
{
    // Calculate the global thread index
    long o = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure that the thread index is within range
    if (o >= nrows) return;

    // Calculate input offset
    long i = o * ncols;
    
    // Initialize minimum and index values
    float min = input[i];
    long argmin = 0;
    
    // Loop unrolling for better performance when ncols is large
    for (long ii = 1; ii < ncols; ii += 4) {
        if (ii + 0 < ncols && input[i + ii + 0] < min) {
            min = input[i + ii + 0];
            argmin = ii + 0;
        }
        if (ii + 1 < ncols && input[i + ii + 1] < min) {
            min = input[i + ii + 1];
            argmin = ii + 1;
        }
        if (ii + 2 < ncols && input[i + ii + 2] < min) {
            min = input[i + ii + 2];
            argmin = ii + 2;
        }
        if (ii + 3 < ncols && input[i + ii + 3] < min) {
            min = input[i + ii + 3];
            argmin = ii + 3;
        }
    }

    // Store results
    output[o] = min;
    indices[o] = argmin + 1;
}