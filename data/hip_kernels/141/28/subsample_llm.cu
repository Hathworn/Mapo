#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;
    
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    output += o * output_w * output_h;
    input += i * input_w * input_h;

    // For all output pixels...
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            // Compute the mean of the input image
            float *ptr_input = input + yy * dH * input_w + xx * dW;
            float *ptr_output = output + yy * output_w + xx;
            float sum = 0;
            
            // Efficiently compute sum using shared loop variable
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    sum += ptr_input[kx];
                }
                ptr_input += input_w; // next input line
            }
            
            // Update output
            *ptr_output = sum / float(kW * kH);
        }
    }
}