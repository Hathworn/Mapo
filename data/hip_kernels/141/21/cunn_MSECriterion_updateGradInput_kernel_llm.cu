#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_MSECriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, float norm, int nframe, int dim)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < nframe * dim) { // Only compute if within bounds
        int k = index / dim;
        int i = index % dim;
        float *gradInput_k = gradInput + k * dim;
        float *input_k = input + k * dim;
        float *target_k = target + k * dim;
        gradInput_k[i] = norm * (input_k[i] - target_k[i]); // Compute gradInput
    }
}