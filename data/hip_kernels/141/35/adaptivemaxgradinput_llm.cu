#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w)
{
    // Compute global thread indices
    int xx = blockIdx.x * blockDim.x + threadIdx.x;
    int yy = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread is within output bounds
    if (xx < output_w && yy < output_h) {
        // Compute the offset for the current output plane
        int o = blockIdx.z;
        gradOutput = gradOutput + o * output_w * output_h;
        gradInput = gradInput + o * input_w * input_h;
        indices_x = indices_x + o * output_w * output_h;
        indices_y = indices_y + o * output_w * output_h;

        // Calculate input indices
        int y_start = (int)floorf(float(yy) / output_h * input_h);
        int x_start = (int)floorf(float(xx) / output_w * input_w);

        // Get pointer to the relevant locations
        float *ptr_gradInput = gradInput + y_start * input_w + x_start;
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float *ptr_ind_x = indices_x + yy * output_w + xx;
        float *ptr_ind_y = indices_y + yy * output_w + xx;

        // Avoid shared memory; Directly update gradInput
        float z = *ptr_gradOutput;
        int argmax_x = (*ptr_ind_x) - 1;
        int argmax_y = (*ptr_ind_y) - 1;

        atomicAdd(&ptr_gradInput[argmax_x + argmax_y * input_w], z);
    }
}