#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void maxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    
    // output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    int xx = threadIdx.x + blockIdx.y * blockDim.x;  // Combine xx_start, yy_start in a single value for better performance
    int yy = threadIdx.y + blockIdx.z * blockDim.y * blockDim.x; 

    // select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // compute gradInput
    while (yy < output_h) {
        while (xx < output_w) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float *ptr_ind_x = indices_x + yy * output_w + xx;
            float *ptr_ind_y = indices_y + yy * output_w + xx;
            float z = *ptr_gradOutput;

            int argmax_x = (*ptr_ind_x) - 1;
            int argmax_y = (*ptr_ind_y) - 1;

            atomicAdd(&ptr_gradInput[argmax_x + argmax_y * input_w], z);  // Use atomicAdd for concurrent updates.
            
            xx += blockDim.x * gridDim.y;
        }
        xx = threadIdx.x + blockIdx.y * blockDim.x;  // Re-initialize xx for the next iteration
        yy += blockDim.y * gridDim.z * blockDim.x;
    }
}