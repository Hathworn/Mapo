#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int xx_end = output_w;
    int yy_end = output_h;
    int xx_step = blockDim.x;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;

    // Get weight
    float the_weight = weight[k];

    // Optimize by reducing redundant calculations
    for (int yy = yy_start; yy < yy_end; yy += yy_step) {
        for (int xx = xx_start; xx < xx_end; xx += xx_step) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput * the_weight;
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    atomicAdd(&ptr_gradInput[kx + ky * input_w], z); // Combine pointer arithmetic
                }
            }
        }
    }
}