#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinput(float *gradInput, float *gradOutput, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx_start = blockIdx.y * blockDim.x + threadIdx.x; // Flatten grid for x
    int yy_start = blockIdx.z * blockDim.y + threadIdx.y; // Flatten grid for y

    int xx_step = blockDim.x * gridDim.y; // Adjust step for x
    int yy_step = blockDim.y * gridDim.z; // Adjust step for y

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;

    // Compute gradInput
    for(int yy = yy_start; yy < output_h; yy += yy_step) {
        for(int xx = xx_start; xx < output_w; xx += xx_step) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput;
            for(int ky = 0; ky < kH; ky++) {
                for(int kx = 0; kx < kW; kx++) {
                    ptr_gradInput[kx] += z / float(kW * kH);
                }
                ptr_gradInput += input_w;
            }
        }
    }
}