#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void max_output(float *input, float *output, float *indices, long nrows, long ncols) {
    // Compute global thread index
    long o = blockIdx.x * blockDim.x + threadIdx.x;
    if (o >= nrows) return;

    // Locate row in input matrix
    long i = o * ncols;

    // Initialize max and argmax
    float max = input[i];
    long argmax = 0;

    // Unroll loop to reduce branches and improve performance
    for (long ii = 1; ii < ncols; ++ii) {
        float val = input[i + ii];
        if (val > max) {
            max = val;
            argmax = ii;
        }
    }

    // Store results
    output[o] = max;
    indices[o] = argmax + 1;
}