#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use cooperative groups for better synchronization
__global__ void max_gradInput(float *input, float *output, float *indices, long nrows, long ncols)
{
    // Calculate output offset using blockIdx.x and threadIdx.x
    long o = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread corresponds to valid row
    if (o < nrows) {
        // Calculate input offset
        long i = o * ncols;

        // Calculate index and update input
        long idx = static_cast<long>(indices[o]) - 1;
        if (idx >= 0 && idx < ncols) {
            // Using atomic operation for potential concurrent writes
            atomicAdd(&input[i + idx], output[o]);
        }
    }
}