#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradweight(float *input, float *gradOutput, float *gradWeight, float *gradBias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW, float scale)
{
    // Compute output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    // Start indices and step sizes for iterating output
    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;

    int yy_start = threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y;

    // Select current gradOutput and input plane
    gradOutput += o * output_w * output_h;
    input += i * input_w * input_h;

    // Thread ID and shared memory for partial sums
    int tid = blockDim.x * threadIdx.y + threadIdx.x;
    __shared__ float sums[CUDA_MAX_THREADS];
    sums[tid] = 0;

    // Compute partial sums
    for (int yy = yy_start; yy < yy_end; yy += yy_step) {
        for (int xx = xx_start; xx < xx_end; xx += xx_step) {
            float *ptr_input = input + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput;
            
            // Unroll innermost loop for better performance
            for (int ky = 0; ky < kH; ++ky) {
                #pragma unroll
                for (int kx = 0; kx < kW; ++kx) {
                    sums[tid] += z * ptr_input[kx];
                }
                ptr_input += input_w;
            }
        }
    }
    __syncthreads();

    // Reduce: accumulate partial sums to 1st thread in block
    if (tid == 0) {
        float gradWeightSum = 0;
        for (int i = 0; i < blockDim.x * blockDim.y; i++) {
            gradWeightSum += sums[i];
        }
        atomicAdd(&gradWeight[k], scale * gradWeightSum);
    }
    __syncthreads();

    // Compute gradBias
    sums[tid] = 0;
    for (int i = tid; i < output_w * output_h; i += blockDim.x * blockDim.y) {
        sums[tid] += gradOutput[i];
    }
    __syncthreads();

    // Reduce gradBias to 1st thread in block
    if (tid == 0) {
        float gradBiasSum = 0;
        for (int i = 0; i < blockDim.x * blockDim.y; i++) {
            gradBiasSum += sums[i];
        }
        atomicAdd(&gradBias[k], scale * gradBiasSum);
    }
}