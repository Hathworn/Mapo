#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output size once
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute global thread indices
    int xx = blockIdx.x * blockDim.x + threadIdx.x;
    int yy = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Compute the offset for the output and input based on blockIdx.x
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    // Get the weight and bias for the current plane
    float the_weight = weight[k];
    float the_bias = bias[k];

    // Check if the thread is within bounds
    if (xx < output_w && yy < output_h) {
        // Calculate input and output pointers
        float *ptr_input = input + i * input_w * input_h + yy * dH * input_w + xx * dW;
        float *ptr_output = output + o * output_w * output_h + yy * output_w + xx;
        
        // Compute the sum
        float sum = 0;
        for (int ky = 0; ky < kH; ky++) {
            for (int kx = 0; kx < kW; kx++) {
                sum += ptr_input[kx];
            }
            ptr_input += input_w; // next input line
        }

        // Store the result in the output
        *ptr_output = the_weight * sum + the_bias;
    }
}