#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) 
{
    // Calculate global index based on block and thread indices
    int globalIndex = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    
    // Return early if the global index exceeds or equals output_w
    if (globalIndex >= output_w) return;

    // Calculate offset for each data pointer
    int outputOffset = blockIdx.x * output_w * input_n + globalIndex * input_n;
    int inputOffset = blockIdx.x * input_w * input_n + globalIndex * input_n * dW;

    gradInput += inputOffset;
    gradOutput += outputOffset;
    indices += outputOffset;

    // Iterate over features and perform atomic addition
    for (int feat = 0; feat < input_n; ++feat) {
        atomicAdd(&gradInput[(int)indices[feat] * input_n + feat], gradOutput[feat]);
    }
}