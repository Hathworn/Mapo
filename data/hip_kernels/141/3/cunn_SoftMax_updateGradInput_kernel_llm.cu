#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_SoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int nframe, int dim)
{
    __shared__ float buffer[SOFTMAX_THREADS];
    int k = blockIdx.x;
    float *gradInput_k = gradInput + k * dim;
    float *output_k = output + k * dim;
    float *gradOutput_k = gradOutput + k * dim;

    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;

    // Compute partial sums and store in shared memory
    float sum = 0;
    for (int i = i_start; i < i_end; i += i_step)
        sum += gradOutput_k[i] * output_k[i];
    buffer[threadIdx.x] = sum;
    __syncthreads();

    // Parallel reduction to sum all elements in buffer
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride)
            buffer[threadIdx.x] += buffer[threadIdx.x + stride];
        __syncthreads();
    }

    // Calculate gradInput using the computed sum
    float sum_k = buffer[0];
    for (int i = i_start; i < i_end; i += i_step)
        gradInput_k[i] = output_k[i] * (gradOutput_k[i] - sum_k);
}