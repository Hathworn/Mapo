#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    // Utilize shared memory to optimize memory access
    extern __shared__ float shared_gradOutput[];
    for (int idx = threadIdx.x; idx < output_w * output_h; idx += blockDim.x) {
        int yy = idx / output_w;
        int xx = idx % output_w;
        shared_gradOutput[idx] = gradOutput[o * output_w * output_h + yy * output_w + xx];
    }
    __syncthreads();

    // Compute gradInput in parallel
    for (int yy = blockIdx.y * blockDim.y + threadIdx.y; yy < output_h; yy += blockDim.y * gridDim.y) {
        for (int xx = threadIdx.x; xx < output_w; xx += blockDim.x) {
            float z = shared_gradOutput[yy * output_w + xx];
            float *ptr_gradInput = gradInput + i * input_w * input_h + yy * dH * input_w + xx * dW;
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    atomicAdd(&(ptr_gradInput[kx]), z / float(kW * kH));
                }
                ptr_gradInput += input_w;
            }
        }
    }
}