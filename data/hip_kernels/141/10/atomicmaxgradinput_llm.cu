#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void atomicmaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW) 
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Calculate offsets based on thread/block ID
    int o = blockIdx.x;

    // Calculate 1D index for the grid
    int idx = blockIdx.y * blockDim.y + threadIdx.y;
    int idy = blockIdx.x * blockDim.x + threadIdx.x;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += o * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // Iterate over the output
    for (int yy = idx; yy < output_h; yy += blockDim.y * gridDim.y) {
        for (int xx = idy; xx < output_w; xx += blockDim.x * gridDim.x) {
            
            // Pointer to current position in gradInput, gradOutput, indices
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float *ptr_ind_x = indices_x + yy * output_w + xx;
            float *ptr_ind_y = indices_y + yy * output_w + xx;
            
            float z = *ptr_gradOutput;
            int argmax_x = (*ptr_ind_x) - 1;
            int argmax_y = (*ptr_ind_y) - 1;
            
            // Atomic add since different threads could update the same variable
            atomicAdd(&(ptr_gradInput[argmax_x + argmax_y * input_w]), z);
        }
    }
}