#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(int* count_d, float* randomnums)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Optimize by calculating indices directly without extra variables
    double x = randomnums[tid * 2];
    double y = randomnums[tid * 2 + 1];
    double z = (x * x) + (y * y);
    
    // Use ternary operator for concise condition assignment
    count_d[tid] = (z <= 1) ? 1 : 0;
}