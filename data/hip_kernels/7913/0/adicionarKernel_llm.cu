#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adicionarKernel(double* resultado, const double* n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;  // Optimize to handle multiple blocks
    double a = 1, b = 0;
    double temp = -4 * a * n[i];  // Simplify delta calculation
    resultado[i] = sqrt(temp) / 2;  // Store the result directly
}