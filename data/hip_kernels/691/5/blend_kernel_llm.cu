#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void blend_kernel(float *A, float *BP, float *M, float *AP, float alpha, int c, int h, int w) {
    // Calculate the global thread index
    int _id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;
    if (_id < c * size) {
        // Unroll calculations for id and dc
        int id = _id % size, dc = _id / size;

        // Use precomputation for array indexing
        float m_val = M[id];
        float weight = m_val < 0.05f ? 0.f : alpha;

        // Optimize memory accesses by reducing array calculations
        int offset = dc * size + id;
        AP[offset] = A[offset] * weight + BP[offset] * (1.f - weight);
    }
}