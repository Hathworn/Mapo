#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void upsample_corr_kernel(int *curr_corr, int *next_corr, int curr_h, int curr_w, int next_h, int next_w)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < next_h * next_w) {
        int next_x = id % next_w, next_y = id / next_w;

        // Precompute ratios and offsets.
        float w_ratio = (float)next_w / curr_w;
        float h_ratio = (float)next_h / curr_h;
        float w_offset = 0.5 / w_ratio;
        float h_offset = 0.5 / h_ratio;

        // Use precomputed offsets.
        int curr_x = (next_x / w_ratio) + w_offset;
        int curr_y = (next_y / h_ratio) + h_offset;

        curr_x = MAX(MIN(curr_x, curr_w-1), 0);
        curr_y = MAX(MIN(curr_y, curr_h-1), 0);

        int curr_id = curr_y * curr_w + curr_x;
        int curr_x2 = curr_corr[2 * curr_id + 0];
        int curr_y2 = curr_corr[2 * curr_id + 1];

        // Use integer arithmetic for next_x2 and next_y2.
        int next_x2 = __float2int_rn(next_x + (curr_x2 - curr_x) * w_ratio);
        int next_y2 = __float2int_rn(next_y + (curr_y2 - curr_y) * h_ratio);

        next_x2 = MAX(MIN(next_x2, next_w-1), 0);
        next_y2 = MAX(MIN(next_y2, next_h-1), 0);

        next_corr[2 * id + 0] = next_x2;
        next_corr[2 * id + 1] = next_y2;
    }
}