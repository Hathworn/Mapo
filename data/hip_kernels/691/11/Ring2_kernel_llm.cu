#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void Ring2_kernel(float *A, float *BP, int *corrAB, int *mask, int *m, int ring, int c, int h, int w)
{
    // Calculate thread index
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;
    
    // Check if index is valid
    if (id1 < size) {
        // Check mask condition
        if (mask[id1] != 0) {
            int y2 = corrAB[2 * id1 + 1], x2 = corrAB[2 * id1 + 0];
            
            // Optimize loop using shared variables
            #pragma unroll
            for (int dx = -ring; dx <= ring; dx++) {
                #pragma unroll
                for (int dy = -ring; dy <= ring; dy++) {
                    int _x2 = x2 + dx, _y2 = y2 + dy;
                    
                    // Ensure coordinates are within bounds
                    if (_x2 >= 0 && _x2 < w && _y2 >= 0 && _y2 < h) {
                        // Update mask matrix
                        m[_y2 * w + _x2] = 1;
                    }
                }
            }
        }
    }
}