#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void histogram_kernel(float *I, float *minI, float *maxI, float *mask, int nbins, int c, int h, int w, float *hist) {
    int _id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;

    if (_id < c * size) {
        int id = _id % size, dc = _id / size;

        // Use shared memory for reduction if feasible
        extern __shared__ float local_hist[];

        // Initialize local histogram
        for (int i = threadIdx.x; i < nbins; i += blockDim.x) {
            local_hist[i] = 0.0f;
        }
        __syncthreads();

        if (mask[id] >= EPS) {
            float val = I[_id];
            float _minI = minI[dc];
            float _maxI = maxI[dc];

            if (_minI == _maxI) {
                _minI -= 1;
                _maxI += 1;
            }

            if (_minI <= val && val <= _maxI) {
                int idx = MIN((val - _minI) / (_maxI - _minI) * nbins, nbins - 1);
                atomicAdd(&local_hist[idx], 1.0f);
            }
        }

        __syncthreads();

        // Merge local histograms to global
        for (int i = threadIdx.x; i < nbins; i += blockDim.x) {
            atomicAdd(&hist[dc * nbins + i], local_hist[i]);
        }
    }
}