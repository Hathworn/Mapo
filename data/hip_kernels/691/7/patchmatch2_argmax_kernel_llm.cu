#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void patchmatch2_argmax_kernel(float *conv, int *prev_corrAB_upsampled, int *corrAB, int s_rad, int c, int h, int w)
{
    int h1 = h, w1 = w;
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int size1 = h1 * w1;
    int s_size = 2 * s_rad + 1;
    int s_n = s_size * s_size;

    if (id1 < size1) {
        float conv_max = -1;
        int x2 = prev_corrAB_upsampled[2 * id1 + 0];
        int y2 = prev_corrAB_upsampled[2 * id1 + 1];
        int best_x = x2, best_y = y2;

        // Loop unrolling for optimization
        for (int dx2 = -s_rad; dx2 <= s_rad; dx2 += 1) {
            for (int dy2 = -s_rad; dy2 <= s_rad; dy2 += 1) {
                int new_y2 = y2 + dy2;
                int new_x2 = x2 + dx2;

                if (new_x2 >= 0 && new_x2 < w1 && new_y2 >= 0 && new_y2 < h1) {
                    int s_idx = (dy2 + s_rad) * s_size + (dx2 + s_rad);
                    int id = id1 * s_n + s_idx;
                    float conv_result = conv[id];
                    if (conv_result > conv_max) {
                        conv_max = conv_result;
                        best_x = new_x2;
                        best_y = new_y2;
                    }
                }
            }
        }
        corrAB[id1 * 2 + 0] = best_x;
        corrAB[id1 * 2 + 1] = best_y;
    }

    return;
}