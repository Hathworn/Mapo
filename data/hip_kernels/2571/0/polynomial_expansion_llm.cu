#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void polynomial_expansion(float* poly, int degree, int n, float* array)
{
    // Calculate global thread index
    int INX = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (INX < n)
    {
        // Use shared memory for polynomial coefficients if they fit, to reduce global memory access time
        extern __shared__ float shared_poly[];
        if (threadIdx.x < degree + 1)
        {
            shared_poly[threadIdx.x] = poly[threadIdx.x];
        }
        __syncthreads();

        float val = 0.0f;
        float exp = 1.0f;
        
        for (int x = 0; x <= degree; ++x)
        {
            // Calculation using shared memory
            val += exp * shared_poly[x];
            exp *= array[INX];
        }

        // Store result back to global memory
        array[INX] = val;
    }
}