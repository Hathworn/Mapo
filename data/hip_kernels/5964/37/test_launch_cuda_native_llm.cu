#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void test_launch_cuda_native(float * scalar, float * vector, int sxy, int sx , int sy , int sz , int stride)
{
    int id0 = threadIdx.x + blockIdx.x * blockDim.x;
    int id1 = threadIdx.y + blockIdx.y * blockDim.y;
    int id2 = threadIdx.z + blockIdx.z * blockDim.z;

    // Check bounds for all dimensions at once
    if (id0 >= sx || id1 >= sy || id2 >= sz) { return; }

    int index = id2 * sxy + id1 * sx + id0;
    
    // Load scalar and vector elements
    float s = scalar[index];
    float v[3];
    #pragma unroll // Unroll loop for better performance
    for (int i = 0; i < 3; ++i) {
        v[i] = vector[index + i * stride];
    }

    printf("Grid point from CUDA %d %d %d     scalar: %f  vector: %f %f %f \n", id0, id1, id2, s, v[0], v[1], v[2]);
}