#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_count(float *prp_0, int sz)
{
    // Calculate global index for 3D grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;

    // Ensure thread index is within bounds
    int index = idz * sz * sz + idy * sz + idx;
    if (idx < sz && idy < sz && idz < sz) {
        prp_0[index] = index;  // Compute global linear index and store
    }
}