#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_one(float * prp_0,int sz)
{
    // Calculate linear thread index
    int linear_index = blockIdx.x * blockDim.x * sz * sz
                     + blockIdx.y * blockDim.y * sz
                     + blockIdx.z * blockDim.z
                     + threadIdx.x * sz * sz
                     + threadIdx.y * sz
                     + threadIdx.z;

    // Assign value only if within valid range
    if (linear_index < sz * sz * sz) {
        prp_0[linear_index] = 1.0f; // Fill with one
    }
}