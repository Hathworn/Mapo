#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
extern "C"
__global__ void add(int n, double *a, double *b, double *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        // Using registers for better performance
        double ai = a[i];
        double bi = b[i];
        sum[i] = ai + bi;
    }
}