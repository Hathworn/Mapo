#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel function with loop unrolling
extern "C"
__global__ void add(int n, float *a, float *b, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;

    for (int idx = i; idx < n; idx += gridSize) {
        sum[idx] = a[idx] + b[idx];
    }
}