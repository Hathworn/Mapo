#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    int p = y * width + x;

    // Initialize shared memory and compute initial min/max
    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        minvals[tx] = FLT_MAX;
        maxvals[tx] = -FLT_MAX;
    }

    // Accumulate min and max values over 16 rows
    for (int ty = 1; ty < 16; ty++) {
        p += width;
        if (x < width) {
            float val = d_Data[p];
            minvals[tx] = min(minvals[tx], val);
            maxvals[tx] = max(maxvals[tx], val);
        }
    }
    __syncthreads();

    // Reduce within block using parallel reduction
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (tx < offset) {
            minvals[tx] = min(minvals[tx], minvals[tx + offset]);
            maxvals[tx] = max(maxvals[tx], maxvals[tx + offset]);
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}