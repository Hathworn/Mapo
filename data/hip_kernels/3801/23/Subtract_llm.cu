#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height) {
    // Use built-in blockDim instead of hard-coded value
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;
    // Remove __syncthreads() as it's unnecessary here
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}