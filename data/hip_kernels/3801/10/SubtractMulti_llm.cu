#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify x calculation using blockDim.x
    const int y = blockIdx.y * blockDim.y + threadIdx.y;  // Simplify y calculation using blockDim.y
    int sz = height * pitch;
    int p = threadIdx.z * sz + y * pitch + x;
    if (x < width && y < height) {
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
    // Removed __syncthreads() since it's unnecessary for this operation
}