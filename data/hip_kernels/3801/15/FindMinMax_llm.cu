#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    const int b = blockDim.x;
    int p = y * pitch + x;

    // Load data into shared memory with boundary check
    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        minvals[tx] = FLT_MAX;  // Initialize to max value for min calc
        maxvals[tx] = FLT_MIN;  // Initialize to min value for max calc
    }

    // Find min and max values across 16 rows
    for (int ty = 1; ty < 16; ty++) {
        p += pitch;
        if (x < width) {
            float val = d_Data[p];
            minvals[tx] = fminf(minvals[tx], val);
            maxvals[tx] = fmaxf(maxvals[tx], val);
        }
    }
    __syncthreads();

    // Reduction within shared memory
    for (int stride = b >> 1; stride > 0; stride >>= 1) {
        if (tx < stride) {
            minvals[tx] = fminf(minvals[tx], minvals[tx + stride]);
            maxvals[tx] = fmaxf(maxvals[tx], maxvals[tx + stride]);
        }
        __syncthreads();
    }

    // Store result from each block's shared memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}