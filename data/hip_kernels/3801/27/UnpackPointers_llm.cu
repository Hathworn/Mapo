#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;

    // Calculate block-wide stride for threads to work on separate y blocks
    const int blockStrideY = gridDim.y * 32;
    const int blockStrideX = blockDim.x * 16;

    // Loop over y and x in a more efficient manner using strides
    for (int y = blockIdx.y * 32; y < h; y += blockStrideY) {
        for (int x = blockIdx.x * blockDim.x * 16; x < w; x += blockStrideX) {
            unsigned int val = minmax[(y/32)*w + x + tx];

            if (val) {
                // Unrolling the loop to handle the val bits could improve efficiency
                for (int k = 0; k < 32 && numPts < maxPts; k++, val >>= 1) {
                    if (val & 0x1) {
                        ptrs[numPts * 16 + tx] = (y + k) * w + x + tx;
                        numPts++;
                    }
                }
            }
        }
    }
}