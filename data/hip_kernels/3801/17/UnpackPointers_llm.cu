#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    // Calculate the column index based on thread and block size
    int numPts = 0;
    for (int y = 0; y < h / 32; y++) {
        // Use shared memory for block-wise column access
        __shared__ unsigned int s_val[32];
        
        for (int x = tx; x < w; x += 16) {
            s_val[tx] = minmax[y * w + x]; // Load to shared memory
            
            // Synchronize and process each thread in block
            __syncthreads();

            unsigned int val = s_val[tx];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        // Atomic add ensures unique indices
                        int index = atomicAdd(ptrs, 1);
                        ptrs[16 * index + tx] = (y * 32 + k) * w + x; 
                        numPts++;
                    }
                    val >>= 1;
                }
            }
            __syncthreads();
        }
    }
}