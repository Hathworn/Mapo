#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h) {
    __shared__ float data[16*15];
    __shared__ float hist[32];
    __shared__ float gauss[16];
    
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    
    // Pre-compute Gaussian weights
    if (tx < 15) {
        float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));
    }

    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;

    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + tx;
        int xi = xp + tx;
        int yi = yp + y;
        xi = max(0, min(xi, w-1));
        yi = max(0, min(yi, h-1));
        
        if (tx >= 0 && tx < 15) {
            data[memPos] = g_Data[yi * w + xi];
        }
    }
    __syncthreads();

    // Initialize histogram
    if (tx < 32) hist[tx] = 0.0f;
    __syncthreads();
    
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + tx;
        if (tx >= 1 && tx < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = 16.0f * atan2f(dy, dx) / 3.1416f + 16.5f;
            bin = (bin == 32) ? 0 : bin;
            float grad = sqrtf(dx * dx + dy * dy);
            
            atomicAdd(&hist[bin], grad * gauss[tx] * gauss[y]);
        }
    }
    __syncthreads();

    // Smooth histogram
    float smoothedHist[32];
    if (tx < 32) {
        float v0 = hist[(tx + 31) & 31];
        float v1 = hist[tx];
        float v2 = hist[(tx + 1) & 31];
        float v3 = hist[(tx + 2) & 31];
        float v_1 = hist[(tx + 30) & 31];
        
        smoothedHist[tx] = 6*v1 + 4*(v0 + v2) + (v_1 + v3);
    }
    __syncthreads();

    // Find local maximums
    float maxval1 = 0.0f, maxval2 = 0.0f;
    int i1 = -1, i2 = -1;

    if (tx == 0) {
        for (int i = 0; i < 32; i++) {
            float v = smoothedHist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }

        float val1 = smoothedHist[(i1 + 1) & 31];
        float val2 = smoothedHist[(i1 + 31) & 31];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Orient[bx] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);

        if (maxval2 < 0.8f * maxval1) {
            i2 = -1;
        }
        if (i2 >= 0) {
            val1 = smoothedHist[(i2 + 1) & 31];
            val2 = smoothedHist[(i2 + 31) & 31];
            peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Orient[bx + maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        } else {
            d_Orient[bx + maxPts] = i2;
        }
    }
}