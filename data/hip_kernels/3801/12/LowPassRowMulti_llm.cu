#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LowPassRowMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    extern __shared__ float data[]; // Dynamic shared memory for flexibility
    const int tx = threadIdx.x;
    const int block = blockIdx.x / (NUM_SCALES + 3);
    const int scale = blockIdx.x - (NUM_SCALES + 3) * block;
    const int xout = block * CONVROW_W + tx;
    const int loadPos = xout - RADIUS;
    const int yptr = blockIdx.y * pitch;
    const int writePos = yptr + height * pitch * scale + xout;
    float *kernel = d_Kernel + scale * 16;

    // Load data into shared memory ensuring data locality
    if (loadPos < 0)
        data[tx] = d_Data[yptr];
    else if (loadPos >= width)
        data[tx] = d_Data[yptr + width - 1];
    else
        data[tx] = d_Data[yptr + loadPos];
    __syncthreads();

    // Perform convolution only if within bounds
    if (xout < width && tx < CONVROW_W) {
        float sum = 0.0f;
        #pragma unroll // Unroll loop for optimization
        for (int i = 0; i < 5; ++i) {
            sum += (data[tx + i] + data[tx + 8 - i]) * kernel[i];
        }
        sum -= data[tx + 8]*kernel[4]; // Adjust sum for the defined kernel calculation
        d_Result[writePos] = sum;
    }
}