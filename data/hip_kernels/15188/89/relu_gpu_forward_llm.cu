#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for efficient memory access
    for (int i = tid; i < N; i += stride) {
        out[i] = fmaxf(in[i], 0.0f); // Use fmaxf for potential performance benefit
    }
}