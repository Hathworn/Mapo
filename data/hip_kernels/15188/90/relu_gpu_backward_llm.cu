#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate global stride
    for (int i = tid; i < N; i += stride) { // Use loop with stride for better utilization
        ingrad[i] = indata[i] > 0 ? outgrad[i] : 0; // Update value conditionally
    }
}