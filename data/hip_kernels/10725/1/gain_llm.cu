#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gain(int width, int height, float rGain, float gGain, float bGain, float* input, float* output)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Optimize by calculating index once and using shared memory for faster access
    if ((x < width) && (y < height))
    {
        int index = (y * width + x) * 4;
        
        // Using registers to optimize arithmetic operations
        float r = input[index + 0];
        float g = input[index + 1];
        float b = input[index + 2];
        float a = input[index + 3];

        output[index + 0] = r * rGain;
        output[index + 1] = g * gGain;
        output[index + 2] = b * bGain;
        output[index + 3] = a;
    }
}