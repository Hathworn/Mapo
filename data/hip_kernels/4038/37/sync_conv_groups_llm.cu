#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups() {
    // Use __syncthreads() to ensure threads within the same block reach synchronization point
    __syncthreads();
}