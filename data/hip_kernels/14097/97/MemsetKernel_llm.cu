#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate the global index instead of individual i and j
    int idx = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * blockIdx.y) + blockIdx.x * blockDim.x * blockDim.y;

    if (idx >= w * h) return;  // Ensure index is within bounds

    image[idx] = value;  // Set the value directly by global index
}