#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;
    
    // Precompute common indices and values to reduce redundancy
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    
    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);

    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);
    
    float value = src[image_row_offset + j];

    // Use check bounds flags to avoid repeated condition checks
    bool valid_tx_ty = !(tx >= w || tx < 0 || ty >= h || ty < 0);
    bool valid_txm1_ty = !(tx-1 >= w || tx-1 < 0 || ty >= h || ty < 0);
    bool valid_txm1_tym1 = !(tx-1 >= w || tx-1 < 0 || ty-1 >= h || ty-1 < 0);
    bool valid_tx_tym1 = !(tx >= w || tx < 0 || ty-1 >= h || ty-1 < 0);

    // Fill pixel containing bottom right corner
    if (valid_tx_ty)
    {
        float weight = dx * dy;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Fill pixel containing bottom left corner
    if (valid_txm1_ty)
    {
        float weight = (1.0f - dx) * dy;
        _atomicAdd(dst + ty * image_stride + (tx - 1), value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + (tx - 1), weight);
    }

    // Fill pixel containing upper left corner
    if (valid_txm1_tym1)
    {
        float weight = (1.0f - dx) * (1.0f - dy);
        _atomicAdd(dst + (ty - 1) * image_stride + (tx - 1), value * weight);
        _atomicAdd(normalization_factor + (ty - 1) * image_stride + (tx - 1), weight);
    }

    // Fill pixel containing upper right corner
    if (valid_tx_tym1)
    {
        float weight = dx * (1.0f - dy);
        _atomicAdd(dst + (ty - 1) * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + (ty - 1) * image_stride + tx, weight);
    }
}