#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate global thread index
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for scaling factor reuse
    __shared__ float shared_scale;

    // Load scale factor into shared memory once
    if (threadIdx.x == 0)
    {
        shared_scale = scale;
    }
    __syncthreads();

    // Perform computation only if within bounds
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];

        // Utilize shared memory scale factor
        x[ptidx] = loc.x * shared_scale;
        y[ptidx] = loc.y * shared_scale;
    }
}