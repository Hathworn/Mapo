#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Load shared variables
    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Calculate position
    float cx = u_ * time_scale + (float)j + 1.0f;
    float cy = v_ * time_scale + (float)i + 1.0f;

    // Convert to integer using round-to-nearest
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Load source value
    float value = src[image_row_offset + j];

    // Perform bounds check before atomic add
    if (tx < w && tx >= 0 && ty < h && ty >= 0)
    {
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}
```
