#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global thread index once
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Early exit for out-of-bounds threads
    if (i >= h || j >= w) return;
    
    // Calculate position index for accessing arrays
    const int pos = i * s + j;

    // Directly use ternary operator for normalization factor scaling
    float invScale = (normalization_factor[pos] == 0.0f) ? 1.0f : (1.0f / normalization_factor[pos]);

    image[pos] *= invScale; // Apply normalization
}