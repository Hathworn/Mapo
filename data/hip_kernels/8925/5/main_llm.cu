#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "find_maximum_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    double *array = NULL;
hipMalloc(&array, XSIZE*YSIZE);
double *max = NULL;
hipMalloc(&max, XSIZE*YSIZE);
int dSize = XSIZE*YSIZE;
int *d_mutex = NULL;
hipMalloc(&d_mutex, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
find_maximum<<<gridBlock, threadBlock>>>(array,max,dSize,d_mutex);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
find_maximum<<<gridBlock, threadBlock>>>(array,max,dSize,d_mutex);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
find_maximum<<<gridBlock, threadBlock>>>(array,max,dSize,d_mutex);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}