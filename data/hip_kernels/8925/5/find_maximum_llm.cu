#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void find_maximum(double *array, double *max, int dSize, int *d_mutex) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    
    __shared__ double cache[threadsPerBlock];
    
    double temp = -DBL_MAX; // Use library constant for clarity.
    for (int i = index; i < dSize; i += stride) {
        temp = fmax(temp, array[i]);  // Simplified loop logic.
    }
    
    cache[threadIdx.x] = temp;
    __syncthreads();

    // Reduction
    for (unsigned int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (threadIdx.x < i) {
            cache[threadIdx.x] = fmax(cache[threadIdx.x], cache[threadIdx.x + i]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        while (atomicCAS(d_mutex, 0, 1) != 0);  // Lock
        *max = fmax(*max, cache[0]);
        atomicExch(d_mutex, 0);  // Unlock
    }
}