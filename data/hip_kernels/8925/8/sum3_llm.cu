#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum3(double *d_result, double *d_a, double *d_b, double *d_c, int dSize) {
    // Use shared memory to optimize data access
    extern __shared__ double shared_mem[];
    double *s_a = &shared_mem[0];
    double *s_b = &shared_mem[blockDim.x];
    double *s_c = &shared_mem[2 * blockDim.x];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Load data into shared memory
    if (tid < dSize) {
        s_a[threadIdx.x] = d_a[tid];
        s_b[threadIdx.x] = d_b[tid];
        s_c[threadIdx.x] = d_c[tid];
    }

    __syncthreads(); // Ensure all loads are complete

    // Compute sum using shared memory
    if (tid < dSize) {
        d_result[tid] = s_a[threadIdx.x] + s_b[threadIdx.x] + s_c[threadIdx.x];
    }
}