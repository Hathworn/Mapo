#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void meanMatrix(double *dMatrix, double *dMean, int dSize, int *d_mutex) {
    __shared__ double cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    double temp = 0;

    // Optimize loop increment for better performance
    for (int i = tid; i < dSize; i += blockDim.x * gridDim.x) {
        temp += dMatrix[i];
    }

    // Store temporary sum in shared memory
    cache[cacheIndex] = temp;
    __syncthreads();

    // Reduce within block using shared memory
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    // Atomic add with mutex lock on cache[0] result
    if (cacheIndex == 0) {
        while (atomicCAS(d_mutex, 0, 1) != 0); // lock
        atomicAdd(dMean, cache[0]);
        atomicExch(d_mutex, 0); // unlock
    }

    // Ensure mean division is complete outside the atomic operation
    if (tid == 0) {
        *dMean = *dMean / dSize;
    }
}