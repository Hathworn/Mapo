#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void applyNormSum(double *dMap, double *dSupFeature, double *dMaxSupFeature, double *dMeanSupFeature, double *dInfFeature, double *dMaxInfFeature, double *dMeanInfFeature, int dSize) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Precompute coeffs outside of loop as they are constant for all elements
    double SupCoeff = (dMaxSupFeature[0] - dMeanSupFeature[0]) * (dMaxSupFeature[0] - dMeanSupFeature[0]);
    double InfCoeff = (dMaxInfFeature[0] - dMeanInfFeature[0]) * (dMaxInfFeature[0] - dMeanInfFeature[0]);

    // Use stride loop for better GPU memory access pattern
    for (int i = tid; i < dSize; i += blockDim.x * gridDim.x) {
        dMap[i] += dSupFeature[i] * SupCoeff + dInfFeature[i] * InfCoeff;
    }
}