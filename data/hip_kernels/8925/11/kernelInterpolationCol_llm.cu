#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelInterpolationCol(double *result, int rows, int cols, int factor) {
    // Define shared memory for row calculations
    extern __shared__ double shared[];

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Use shared memory to store column results for faster access
    if (threadIdx.x < cols * factor && threadIdx.y < rows) {
        int trueY = y * factor;
        int offset = x + trueY * cols * factor;

        double a = result[offset];
        double b = result[offset + cols * factor * factor];
        double m = (b - a) / ((double)factor);

        // Interpolation loop with shared memory optimization
        if (y != rows - 1) {
            for (int p = 0; p <= factor; ++p) {
                shared[threadIdx.y * blockDim.x + threadIdx.x] = a;
                a += m;
                offset += cols * factor * factor;
            }
        } else {
            for (int p = 0; p < factor; ++p) {
                shared[threadIdx.y * blockDim.x + threadIdx.x] = b;
                b -= m;
                offset += cols * factor * factor;
            }
        }

        // Write results back to global memory
        for (int p = 0; p < factor; ++p) {
            result[offset] = shared[threadIdx.y * blockDim.x + threadIdx.x];
        }
    }
}