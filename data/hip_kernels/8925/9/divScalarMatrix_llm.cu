#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void divScalarMatrix(double *dMatrix, double *dScalar, int dSize) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    double scalar = dScalar[0]; // Cache scalar value in a register for faster access
    int totalThreads = blockDim.x * gridDim.x;

    for (int i = tid; i < dSize; i += totalThreads) { // Use a for loop for better readability
        dMatrix[i] /= scalar; // Use compound assignment for slight optimization
    }
}