#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void absDifference(double *dDifference, double *dSup, double *dLow, int dSize) {
    // Calculate global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use stride-based loop; stride is the total number of threads
    int stride = blockDim.x * gridDim.x;
    
    for (int i = tid; i < dSize; i += stride) {
        double a = dSup[i];
        double b = dLow[i];
        dDifference[i] = fabs(a - b); // Use built-in function for absolute value
    }
}