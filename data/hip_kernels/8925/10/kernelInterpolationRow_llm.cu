#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelInterpolationRow(double *original, double *result, int rows, int cols, int factor) {
    // Calculate unique index for each thread
    int x = blockIdx.y * blockDim.y + threadIdx.y;
    int y = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate original and result indices
    int idOriginal, idResult;

    // Reference points for interpolation
    double a, b;
    double m;

    // Perform row-wise interpolation
    if (x < rows - 1 && y < cols) {
        idOriginal = y * rows + x;
        idResult = y * rows * factor * factor + x * factor;

        // Load values from global memory
        a = original[idOriginal];
        b = original[idOriginal + 1];

        // Calculate interpolation step
        m = (b - a) / ((double)factor);

        // Before reaching the end
        for (int p = 0; p <= factor; ++p) {
            if (p < factor) {  // Avoid out-of-bounds
                result[idResult + p] = a;
                a += m;
            }
        }
    } 

    // Handle the last row separately if needed
    else if (x == rows - 1 && y < cols) {
        idOriginal = y * rows + x;
        idResult = y * rows * factor * factor + x * factor;

        // Load the last value
        b = original[idOriginal];

        // Final border processing
        for (int p = 0; p < factor; ++p) {
            result[idResult + p] = b;
        }
    }
}
```
