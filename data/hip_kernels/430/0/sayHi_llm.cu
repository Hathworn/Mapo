#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use __constant__ memory if necessary or optimize memory usage
__global__ void sayHi()
{
    // Utilize warp shuffle for efficient communication within thread block if needed
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID

    if (tid == 0) {
        // Print only once per kernel launch
        printf("Cuda Kernel Hello World.\n");
    }
}