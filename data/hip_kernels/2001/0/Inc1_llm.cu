#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

// Optimized kernel function
__global__ void Inc1(float *Ad, float *Bd){
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    // Check for bounds and limit the active thread
    if(tx == 0){
        float a = Ad[tx]; // Load value once to register
        for(int i = 0; i < ITER; i++){
            a += 1.0f; // Increment in register
            // Reduce unnecessary access by using a register variable
            float temp = a;
            for(int j = 0; j < 256; j++){
                Bd[tx] = temp;
            }
        }
        Ad[tx] = a; // Store the incremented value back to memory once
    }
}