#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function for matrix multiplication
__global__ void matrixMulCUDA1(float *C, float *A, float *B, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // Calculate global row index
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global column index
    
    if (row < n && col < n) { // Boundary check to prevent out of bounds access
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) { 
            sum += A[row * n + k] * B[k * n + col];  // Perform dot product
        }
        C[row * n + col] = sum; // Store the result
    }
}