#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define TILE_WIDTH 16

__global__ void matrixMulCUDA3(float *C, float *A, float *B, int n) {
    // Use shared memory to reduce global memory accesses
    __shared__ float As[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    float C_val = 0;

    for (int k = 0; k < (n + TILE_WIDTH - 1) / TILE_WIDTH; ++k) {
        // Load data into shared memory, check bounds
        if (row < n && k * TILE_WIDTH + threadIdx.x < n)
            As[threadIdx.y][threadIdx.x] = A[row * n + k * TILE_WIDTH + threadIdx.x];
        else
            As[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < n && k * TILE_WIDTH + threadIdx.y < n)
            Bs[threadIdx.y][threadIdx.x] = B[(k * TILE_WIDTH + threadIdx.y) * n + col];
        else
            Bs[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        // Compute dot product for one tile
        for (int e = 0; e < TILE_WIDTH; ++e)
            C_val += As[threadIdx.y][e] * Bs[e][threadIdx.x];

        __syncthreads();
    }

    // Write the result
    if (row < n && col < n)
        C[row * n + col] = C_val;
}