#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMulCUDA2(float *C, float *A, float *B, int n) {
    // Calculate the row and column indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check to avoid accessing outside of matrix boundaries
    if (row < n && col < n) {
        float C_val = 0;

        // Use shared memory for better cache utilization
        extern __shared__ float sharedMem[];
        float* As = sharedMem;
        float* Bs = As + blockDim.x * blockDim.y;

        // Loop through tiles
        for (int tile = 0; tile < (n + blockDim.x - 1) / blockDim.x; ++tile) {
            // Load elements into shared memory
            if (tile * blockDim.x + threadIdx.x < n && row < n) {
                As[threadIdx.y * blockDim.x + threadIdx.x] = A[row * n + tile * blockDim.x + threadIdx.x];
            } else {
                As[threadIdx.y * blockDim.x + threadIdx.x] = 0.0f;
            }
            if (tile * blockDim.y + threadIdx.y < n && col < n) {
                Bs[threadIdx.y * blockDim.x + threadIdx.x] = B[(tile * blockDim.y + threadIdx.y) * n + col];
            } else {
                Bs[threadIdx.y * blockDim.x + threadIdx.x] = 0.0f;
            }

            // Synchronize before computation
            __syncthreads();

            // Perform partial matrix multiplication
            for (int k = 0; k < blockDim.x; ++k) {
                C_val += As[threadIdx.y * blockDim.x + k] * Bs[k * blockDim.x + threadIdx.x];
            }

            // Synchronize before loading the next tile
            __syncthreads();
        }

        // Write the result back to global memory
        C[row * n + col] = C_val;
    }
}