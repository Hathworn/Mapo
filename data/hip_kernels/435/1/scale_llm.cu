#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 512

__global__ void scale(double *d, int m, int ld, double *s) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < m) {
        // Cache the square root of s to avoid repeated calculation
        double scale_factor = rsqrt(*s);
        d[idx * ld] *= scale_factor; // Applying scale
    }
}