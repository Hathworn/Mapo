#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 512

__global__ void r1_update(double *A, int m, int n, int lda, double *col, int ldc, double *row) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Optimize loop to minimize redundant computation
    if (idx < m && idy < n - 1) {
        double col_val = col[idy * ldc]; // Load once per idy
        for (int ii = 0; ii < n - 1; ii++) {
            A[idx * lda + ii + 1] -= col_val * row[ii + 1];
        }
    }
}