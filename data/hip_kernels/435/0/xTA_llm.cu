#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define THREADS_PER_BLOCK 512

__global__ void xTA(double *y, int k, double *A, int m, int lda, double *x, int ldx) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < k) {
        double s = 0.0; // Initialize sum to 0
        for (int ii = 0; ii < m; ii++) {
            s += x[ii * ldx] * A[idx + ii * lda];
        }
        y[idx] = s; // Update result vector 
    }
}