#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void count(int *A, int *B, int n) {
    
    // Calculate block and thread specific parameters
    int b_id = blockIdx.x;
    int t_id = threadIdx.x;
    int t_num = blockDim.x;
    
    // Declare shared memory for counting
    __shared__ int count[MAX_VALUE];
    
    // Initialize shared memory counts to zero
    for (int i = t_id; i < MAX_VALUE; i += t_num) {
        count[i] = 0;
    }
    
    __syncthreads(); // Synchronize after initialization
    
    // Calculate the portion each thread will handle
    int b_size = (n + gridDim.x - 1) / gridDim.x; // ceil(n / b_num)
    int b_offset = b_id * b_size;
    
    for (int i = b_offset + t_id; i < n && i < b_offset + b_size; i += t_num) {
        atomicAdd(&count[A[i]], 1);
    }
    
    __syncthreads(); // Synchronize after counting phase
    
    // Copy counts from shared memory to global memory
    for (int i = t_id; i < MAX_VALUE; i += t_num) {
        B[b_id * MAX_VALUE + i] = count[i];
    }

}