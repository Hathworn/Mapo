#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}

__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);

    // Precompute start, middle, and end for all slices
    long start = width * idx * slices;
    long offsets[2] = {0, width >> 1};  // Precompute offsets for start and middle

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size) break;

        // Calculate middle and end
        long middle = min(start + offsets[1], size);
        long end = min(start + width, size);

        // Perform merge
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}