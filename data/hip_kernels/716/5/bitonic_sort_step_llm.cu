#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bitonic_sort_step(int *dev_values, int j, int k)
{
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ixj = i ^ j;

    // Ensure valid pair
    if (ixj > i) {
        int temp;
        // Shared memory for faster access
        __shared__ int shared_data[1024]; // assuming a maximum of 1024 threads
        shared_data[threadIdx.x] = dev_values[i];
        shared_data[threadIdx.x + (blockDim.x / 2)] = dev_values[ixj];
        __syncthreads();

        // Ascending or descending sort
        bool swap_flag = ((i & k) == 0 && shared_data[threadIdx.x] > shared_data[threadIdx.x + (blockDim.x / 2)]) ||
                         ((i & k) != 0 && shared_data[threadIdx.x] < shared_data[threadIdx.x + (blockDim.x / 2)]);

        if (swap_flag) {
            temp = shared_data[threadIdx.x];
            shared_data[threadIdx.x] = shared_data[threadIdx.x + (blockDim.x / 2)];
            shared_data[threadIdx.x + (blockDim.x / 2)] = temp;
        }

        __syncthreads();
        // Write sorted data back to global memory
        dev_values[i] = shared_data[threadIdx.x];
        dev_values[ixj] = shared_data[threadIdx.x + (blockDim.x / 2)];
    }
}