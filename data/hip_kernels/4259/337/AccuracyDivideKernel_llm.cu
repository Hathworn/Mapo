#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use the first thread only as this is a scalar operation
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}