#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(int *a, int *b, int *c) {
    // Use shared memory to optimize memory access patterns
    __shared__ int tempA[blockDim.x];
    __shared__ int tempB[blockDim.x];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    while (tid < D) {
        // Load data into shared memory
        tempA[threadIdx.x] = a[tid];
        tempB[threadIdx.x] = b[tid];

        __syncthreads(); // Ensure all loads are complete before proceeding

        // Perform vector addition
        c[tid] = tempA[threadIdx.x] + tempB[threadIdx.x];
        
        tid += blockDim.x * gridDim.x;
    }
}