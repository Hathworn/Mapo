#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mikernel(void) {
    // Use blockIdx and threadIdx for conditional execution to reduce redundant calls
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Llamada a kernel de gpu\n");
    }
}