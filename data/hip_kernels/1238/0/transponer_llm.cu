#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transponer(float* entrada, float* salida, int ANCHO) {
    // Calculate unique thread identifiers
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds to avoid out-of-bounds memory access
    if (tx < ANCHO && ty < ANCHO) {
        // Efficiently transpose using shared memory if possible (depends on hardware capabilities)
        salida[tx * ANCHO + ty] = entrada[ty * ANCHO + tx];
    }
}