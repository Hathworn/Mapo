#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// GPU constant memory to hold our kernels (extremely fast access time)
__constant__ float convolutionKernelStore[256];

/**
* Convolution function for cuda.  Destination is expected to have the same width/height as source, but there will be a border
* of floor(kWidth/2) pixels left and right and floor(kHeight/2) pixels top and bottom
*
* @param source      Source image host pinned memory pointer
* @param width       Source image width
* @param height      Source image height
* @param paddingX    source image padding along x
* @param paddingY    source image padding along y
* @param kOffset     offset into kernel store constant memory
* @param kWidth      kernel width
* @param kHeight     kernel height
* @param destination Destination image host pinned memory pointer
*/

// Converts the Pythagorean theorem along a vector on the GPU

// Create an image buffer. Return host ptr, pass out device pointer through pointer to pointer
__global__ void pythagoras(unsigned char *a, unsigned char *b, unsigned char *c)
{
    // Calculate unique global index for each thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use fast math function for better GPU performance
    float af = static_cast<float>(a[idx]);
    float bf = static_cast<float>(b[idx]);

    // Calculate the result using sqrtf
    c[idx] = static_cast<unsigned char>(__fsqrt_rn(af * af + bf * bf));
}