#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// GPU constant memory to hold our kernels (extremely fast access time)
__constant__ float convolutionKernelStore[256];

/**
* Convolution function for cuda. Destination is expected to have the same width/height as source, but there will be a border
* of floor(kWidth/2) pixels left and right and floor(kHeight/2) pixels top and bottom
*
* @param source      Source image host pinned memory pointer
* @param width       Source image width
* @param height      Source image height
* @param paddingX    source image padding along x
* @param paddingY    source image padding along y
* @param kOffset     offset into kernel store constant memory
* @param kWidth      kernel width
* @param kHeight     kernel height
* @param destination Destination image host pinned memory pointer
*/

__global__ void convolve(unsigned char *source, int width, int height, int paddingX, int paddingY, ssize_t kOffset, int kWidth, int kHeight, unsigned char *destination)
{
    // Calculate our pixel's location
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Shared memory for tiles, loads a block of source image into shared memory
    extern __shared__ float shared_source[];
    
    int pWidth = kWidth / 2;
    int pHeight = kHeight / 2;
    
    // Load data into shared memory
    int shared_x = threadIdx.x + pWidth;
    int shared_y = threadIdx.y + pHeight;

    if (x < width && y < height)
    {
        shared_source[shared_y * (blockDim.x + kWidth) + shared_x] = float(source[y * width + x]);
    }

    // Load halo data into shared memory
    if (threadIdx.x < pWidth && x >= pWidth)
    {
        shared_source[shared_y * (blockDim.x + kWidth) + threadIdx.x] = float(source[y * width + x - pWidth]);
    }

    if (threadIdx.x >= blockDim.x - pWidth && x < width - pWidth)
    {
        shared_source[shared_y * (blockDim.x + kWidth) + threadIdx.x + 2 * pWidth] = float(source[y * width + x + pWidth]);
    }

    if (threadIdx.y < pHeight && y >= pHeight)
    {
        shared_source[threadIdx.y * (blockDim.x + kWidth) + shared_x] = float(source[(y - pHeight) * width + x]);
    }

    if (threadIdx.y >= blockDim.y - pHeight && y < height - pHeight)
    {
        shared_source[(threadIdx.y + 2 * pHeight) * (blockDim.x + kWidth) + shared_x] = float(source[(y + pHeight) * width + x]);
    }

    __syncthreads();

    float sum = 0.0;

    // Only execute for valid pixels
    if (x >= pWidth + paddingX &&
        y >= pHeight + paddingY &&
        x < (blockDim.x * gridDim.x) - pWidth - paddingX &&
        y < (blockDim.y * gridDim.y) - pHeight - paddingY)
    {
        for (int j = -pHeight; j <= pHeight; j++)
        {
            for (int i = -pWidth; i <= pWidth; i++)
            {
                // Sample the weight for this location
                int ki = (i + pWidth);
                int kj = (j + pHeight);
                float w = convolutionKernelStore[(kj * kWidth) + ki + kOffset];

                sum += w * shared_source[(shared_y + j) * (blockDim.x + kWidth) + (shared_x + i)];
            }
        }
    }

    // Write result
    if (x < width && y < height) 
    {
        destination[(y * width) + x] = (unsigned char)sum;
    }
}