#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
    // Calculate global index for thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure work is done only within array bounds
    if(i < N)
    {
        c[i] = a[i] + b[i];
    }
}