#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel launch for better occupancy and warp utilization
__global__ void lyra2Z_gpu_hash_32_sm2(uint32_t threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *resNonces) {
    // Calculate the thread index
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread does not work if beyond thread limit
    if (idx >= threads) return;
    
    // Shared memory for intermediate computations to reduce global memory access
    __shared__ uint64_t sharedData[32];
    
    // Load initial hash value into shared memory and synchronize threads
    sharedData[threadIdx.x] = g_hash[idx];
    __syncthreads();
    
    // Compute next hash value using shared memory to lower global memory access latency
    uint64_t newHash = sharedData[threadIdx.x] ^ startNounce;
    
    // Write result to global memory, use atomic to avoid race condition
    atomicAdd(&resNonces[idx], newHash % threads);
}