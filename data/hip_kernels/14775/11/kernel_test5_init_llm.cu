#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_init(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Use early exit to avoid further computation if not needed
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int p1 = 1;
    // Optimize loop unrolling by reducing unnecessary calculations
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i += 16) {
        unsigned int p2 = ~p1;

        ptr[i] = ptr[i+1] = ptr[i+4] = ptr[i+5] = ptr[i+8] = ptr[i+9] = ptr[i+12] = ptr[i+13] = p1;
        ptr[i+2] = ptr[i+3] = ptr[i+6] = ptr[i+7] = ptr[i+10] = ptr[i+11] = ptr[i+14] = ptr[i+15] = p2;

        p1 <<= 1;
        if (p1 == 0){
            p1 = 1;
        }
    }
}