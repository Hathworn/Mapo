#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    // Compute the starting pointer for the block
    unsigned int* orig_ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* ptr = orig_ptr;
    
    // Check if the starting pointer is out of bounds
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Compute the end pointer for the block
    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);
    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Write the initial pattern
    *ptr = pattern;

    // Loop through the block with optimized conditions
    while (true) {
        // Calculate the next address based on the mask
        ptr = (unsigned int*)(((unsigned long)orig_ptr) | mask);
        
        // Break if the mask calculation overflows past block_end
        if (ptr >= block_end) {
            break;
        }

        // Only increment mask at each successful write site
        *ptr = pattern;
        pattern <<= 1;  // Use compound assignment for pattern
        mask <<= 1;     // Increment the mask
    }
    return;
}