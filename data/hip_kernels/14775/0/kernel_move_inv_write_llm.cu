#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    // Calculate the global thread index
    unsigned int idx = blockIdx.x * BLOCKSIZE / sizeof(unsigned int) + threadIdx.x;
    unsigned int* ptr = (unsigned int*)_ptr + idx;
    
    // Check bounds only once before writing pattern
    if (ptr < (unsigned int*) end_ptr) {
        *ptr = pattern;
    }
}
```
