#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Check bounds and exit early if out of range
    if (ptr >= (unsigned long*) end_ptr) {
        return;
    }

    // Utilize threadIdx for parallel writing and avoid redundant computation
    unsigned int tid = threadIdx.x;
    if (tid < BLOCKSIZE / sizeof(unsigned long)) {
        ptr[tid] = (unsigned long) &ptr[tid];
    }
}