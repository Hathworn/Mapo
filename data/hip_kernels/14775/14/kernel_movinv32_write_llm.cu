#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    // Calculate thread-specific pointer offset
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE) + threadIdx.x;
    unsigned int stride = blockDim.x;

    // Return if starting ptr goes out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int k = offset + threadIdx.x;
    unsigned int pat = pattern;

    // Loop considering thread stride
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += stride) {
        // Writing pattern into memory
        ptr[i] = pat;
        k++;
        // Update pattern with wrap-around logic
        if (k >= 32) {
            k = 0;
            pat = lb;
        } else {
            pat = (pat << 1) | sval;
        }
    }
    return;
}