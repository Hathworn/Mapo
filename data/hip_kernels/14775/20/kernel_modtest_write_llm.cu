#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    // Pre-calculate commonly used value
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int blockLimit = BLOCKSIZE / sizeof(unsigned int);

    // Return early if ptr is out of bounds
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Utilize shared memory for improved memory access pattern efficiency
    __shared__ unsigned int sharedMemory[BLOCKSIZE / sizeof(unsigned int)];
    
    // Initialize shared memory with current global memory values
    for (unsigned int i = threadIdx.x; i < blockLimit; i += blockDim.x) {
        sharedMemory[i] = ptr[i];
    }
    __syncthreads();

    // First loop: write p1 to specific positions
    for (unsigned int i = offset + threadIdx.x; i < blockLimit; i += MOD_SZ * blockDim.x) {
        sharedMemory[i] = p1;
    }
    
    // Second loop: write p2 to other positions
    for (unsigned int i = threadIdx.x; i < blockLimit; i += blockDim.x) {
        if (i % MOD_SZ != offset) {
            sharedMemory[i] = p2;
        }
    }
    __syncthreads();

    // Write back results to global memory
    for (unsigned int i = threadIdx.x; i < blockLimit; i += blockDim.x) {
        ptr[i] = sharedMemory[i];
    }
}