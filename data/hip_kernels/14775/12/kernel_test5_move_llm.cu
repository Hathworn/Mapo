#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    unsigned int i;
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);

    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Use half_count variable only once and store computation result
    const unsigned int half_count = BLOCKSIZE / (2 * sizeof(unsigned int));
    unsigned int* ptr_mid = ptr + half_count;

    // Unroll loop for copying data to ptr_mid
    #pragma unroll
    for (i = 0; i < half_count; i++) {
        ptr_mid[i] = ptr[i];
    }

    // Unroll loop for copying data back to ptr with an offset
    #pragma unroll
    for (i = 0; i < half_count - 8; i++) {
        ptr[i + 8] = ptr_mid[i];
    }

    // Unroll loop for handling the first 8 elements
    #pragma unroll
    for (i = 0; i < 8; i++) {
        ptr[i] = ptr_mid[half_count - 8 + i];
    }

    return;
}