#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    // Typecast pointers for easier manipulation
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int* orig_ptr = ptr;

    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Initial write of pattern
    *ptr = pattern;

    // Loop optimized for clarity and efficiency
    while(ptr < end_ptr){
        ptr = (unsigned int*) (((unsigned long)orig_ptr) | mask);
        
        // Optimize loop exit condition handling
        if (ptr >= end_ptr) {
            break;
        }

        // Avoid writing again to the same location
        if (ptr != orig_ptr) {
            *ptr = pattern;
            pattern = pattern << 1;  // Update pattern only when writing to new address
        }
        
        mask = mask << 1;  // Increment mask
    }
    return;
}