#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_reset_draw_counters() {
    if (threadIdx.x == 0 && blockIdx.x == 0) { // Only first thread of first block resets the counter
        r_draw_counter = 0;
    }
}