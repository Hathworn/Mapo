#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function for initializing cells
__global__ void initialize_cells(CellT* __restrict__ dev_cells, CellT* __restrict__ dev_next_cells, int size_x, int size_y) {
    // Calculate the total number of threads
    int totalThreads = blockDim.x * gridDim.x;
    // Calculate the global thread index
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Use a more efficient looping strategy with stride equal to total number of threads
    for (int i = idx; i < size_x * size_y; i += totalThreads) {
        dev_cells[i] = 0;
        dev_next_cells[i] = 0;
    }
}