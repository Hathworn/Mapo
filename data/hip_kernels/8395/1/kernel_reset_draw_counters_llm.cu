#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_reset_draw_counters() {
    // Use atomic operation to reset counter to avoid race conditions
    atomicExch(&r_draw_counter, 0);
}