#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smoothColor(unsigned char *imagem, unsigned char *saida, unsigned int cols, unsigned int linhas) 
{
    unsigned int indice = blockIdx.y * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x; // corrected index calculation
    if (indice >= cols * linhas)
        return;

    // Compute starting and ending indices for the smoothing window
    int i = indice / cols;
    int j = indice % cols;

    int i_begin = max(i - 2, 0);
    int i_end = min(i + 2, linhas - 1);
    int j_begin = max(j - 2, 0);
    int j_end = min(j + 2, cols - 1);

    // Initialize smoothing variables
    int media[3] = {0, 0, 0};
    int qtd = 0;

    for (int ii = i_begin; ii <= i_end; ++ii) 
    {
        for (int jj = j_begin; jj <= j_end; ++jj) 
        {
            // Accumulate pixel values
            media[0] += imagem[((ii * cols) + jj) * 3];
            media[1] += imagem[((ii * cols) + jj) * 3 + 1];
            media[2] += imagem[((ii * cols) + jj) * 3 + 2];
            qtd++;
        }
    }

    // Compute the average and store in the output
    saida[indice * 3] = (unsigned char)(media[0] / qtd);
    saida[indice * 3 + 1] = (unsigned char)(media[1] / qtd);
    saida[indice * 3 + 2] = (unsigned char)(media[2] / qtd);
}