#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smoothGray(unsigned char *imagem, unsigned char *saida, unsigned int cols, unsigned int linhas)
{
    // Calculate the linear index for the current thread
    unsigned int indice = (blockIdx.y * blockDim.y * cols) + (blockIdx.x * blockDim.x) + threadIdx.x;
    if (indice >= cols * linhas)
        return;

    // Calculate bounds for the smoothing window
    int i_begin = (indice / cols) - 2;
    int i_end = (indice / cols) + 2;
    int j_begin = (indice % cols) - 2;
    int j_end = (indice % cols) + 2;

    // Ensure indices are within the image boundaries
    i_begin = max(i_begin, 0);
    j_begin = max(j_begin, 0);
    i_end = min(i_end, linhas - 1);  // Correct the boundary check
    j_end = min(j_end, cols - 1);

    // Calculate the average value in the window
    int media = 0;
    int qtd = 0;
    for (int i = i_begin; i <= i_end; ++i)
    {
        for (int j = j_begin; j <= j_end; ++j)
        {
            media += imagem[(i * cols) + j];
            qtd++;
        }
    }
    saida[indice] = (unsigned char)(media / qtd);
}