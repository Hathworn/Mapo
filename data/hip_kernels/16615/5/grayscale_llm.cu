#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void grayscale(unsigned char *data_rgb, unsigned char *data_gray, std::size_t rows, std::size_t cols)
{
    // Calculate global thread index for both dimensions
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds to ensure indices are within image dimensions
    if (i < cols && j < rows)
    {
        // Calculate single index to avoid redundant calculations
        std::size_t index = j * cols + i;
        // Apply formula to convert RGB to grayscale using predefined constants
        unsigned char gray_value = (
            307 * data_rgb[3 * index] +
            604 * data_rgb[3 * index + 1] +
            113 * data_rgb[3 * index + 2]
        ) / 1024;
        // Assign computed gray value to the output array
        data_gray[index] = gray_value;
    }
}