#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void grayscale(unsigned char * data_rgb, unsigned char * data_gray, std::size_t rows, std::size_t cols)
{
    // Use shared memory to reduce global memory accesses
    extern __shared__ unsigned char shared_rgb[];

    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    // Linear index for the current thread within the block
    int localIdx = threadIdx.y * blockDim.x + threadIdx.x;

    // Load RGB data into shared memory
    if (i < cols && j < rows) {
        shared_rgb[3 * localIdx] = data_rgb[3 * (j * cols + i)];
        shared_rgb[3 * localIdx + 1] = data_rgb[3 * (j * cols + i) + 1];
        shared_rgb[3 * localIdx + 2] = data_rgb[3 * (j * cols + i) + 2];
        __syncthreads(); // Synchronize to ensure all threads have loaded their data

        // Compute grayscale value using loaded shared memory
        data_gray[j * cols + i] = (
            307 * shared_rgb[3 * localIdx] + 
            604 * shared_rgb[3 * localIdx + 1] + 
            113 * shared_rgb[3 * localIdx + 2]
        ) / 1024;
    }
}