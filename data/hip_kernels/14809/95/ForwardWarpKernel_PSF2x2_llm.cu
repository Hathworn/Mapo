#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate coordinates and weight once, reuse them
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    int tx = (int) px;
    int ty = (int) py;
    float value = src[image_row_offset + j];

    #pragma unroll 4 // Unroll the following 4 operations for better performance
    for (int k = 0; k < 4; ++k) {
        // Check bounds and calculate weight
        if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0))) {
            float weight = ((k & 1) ? dx : (1.0f - dx)) * ((k & 2) ? dy : (1.0f - dy));
            _atomicAdd(dst + ty * image_stride + tx, value * weight);
            _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
        }
        // Update coordinates for next corner
        tx += (k == 0) - (k == 1);
        ty += (k == 2) - (k == 3);
    }
}