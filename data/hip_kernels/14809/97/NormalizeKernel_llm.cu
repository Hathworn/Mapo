#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global thread index for 1D array processing
    int pos = (blockIdx.y * blockDim.y + threadIdx.y) * s + blockIdx.x * blockDim.x + threadIdx.x;

    if (pos >= h * s || pos % s >= w) return; // Boundary check
    
    float scale = normalization_factor[pos];
    float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale); // Avoid division by zero
    image[pos] *= invScale;
}