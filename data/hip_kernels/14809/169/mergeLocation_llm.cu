#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate the unique thread index
    int ptidx = blockIdx.x * blockDim.x + threadIdx.x;
   
    // Use a loop to process multiple points per thread to maximize warp efficiency
    for (; ptidx < npoints; ptidx += blockDim.x * gridDim.x)
    {
        short2 loc = loc_[ptidx];
        
        // Perform scaled assignment
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}