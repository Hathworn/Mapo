#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for coalesced access
    // Loop over elements
    for (int i = tid; i < N; i += stride) {
        // Perform ReLU in a loop for full utilization
        out[i] = in[i] > 0 ? in[i] : 0;
    }
}