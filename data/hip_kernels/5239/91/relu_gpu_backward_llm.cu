#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel with improved memory coalescing and simplified computation
__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a loop to handle cases where total threads can be greater than N
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        ingrad[i] = indata[i] > 0 ? outgrad[i] : 0;
    }
}