#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tanh(float *inout, float *bias, int rows, int cols) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Use if statement to ensure only valid threads perform operations
    if (j < cols && i < rows) {
        float value = inout[i * cols + j];
        inout[i * cols + j] = tanhf(value) + bias[i];
    }
}