#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu(float *inout, float *bias, int rows, int cols) {
    // Calculate the global index for the flattened grid
    int idx = blockIdx.y * blockDim.y * cols + blockIdx.x * blockDim.x + threadIdx.y * cols + threadIdx.x;

    // Total number of elements in the 2D grid
    int total_elements = rows * cols;

    // Iterate over the grid in a linear stride pattern
    for (; idx < total_elements; idx += blockDim.x * gridDim.x * blockDim.y * gridDim.y) {
        int i = idx / cols;  // Row index
        int j = idx % cols;  // Column index
        inout[idx] = fmaxf(0.0, inout[idx] + bias[i]);
    }
}