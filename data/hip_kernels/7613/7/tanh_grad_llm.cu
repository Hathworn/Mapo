#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tanh_grad(float *pre_grad, float *output, int rows, int cols) {
    // Compute global thread index for 1D array access
    int index = blockIdx.y * blockDim.y * cols + blockIdx.x * blockDim.x + threadIdx.y * cols + threadIdx.x;

    // Linearize the 2D grid and block structure to a single index
    int total_threads = gridDim.x * blockDim.x * gridDim.y * blockDim.y;
    
    // Loop unrolling in case of insufficient threads, to cover all elements
    for(int idx = index; idx < rows * cols; idx += total_threads) {
        float t = output[idx];
        pre_grad[idx] *= 1 - t * t;
    }
}