#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixTransKernel(float *A, int rows, int cols) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure each thread swaps only elements in the upper triangular part
    if (j > i && j < cols && i < rows) {
        float tmp = A[i * cols + j];
        A[i * cols + j] = A[j * cols + i];
        A[j * cols + i] = tmp;
    }
}