#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void attentionKernel(float *x, int rows, int cols) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (j >= cols) return;

    float sum = 0;
    
    // Use shared memory to reduce global memory access
    extern __shared__ float sharedSum[];
    
    // Step 1: Calculate sum
    for (int k = 0; k < rows; k++) {
        sum += x[k * cols + j];
    }

    // Write sum to shared memory
    sharedSum[threadIdx.x] = sum;
    __syncthreads();

    // Read sum from shared memory reducing calculation overhead
    sum = sharedSum[threadIdx.x];

    // Step 2: Update x using computed sum
    for (int k = 0; k < rows; k++) {
        x[k * cols + j] *= sum;
    }
}