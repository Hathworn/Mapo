#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sigmoid_grad(float *pre_grad, float *output, int rows, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // 1D index for global memory access
    int total_elements = rows * cols;
    
    if (idx < total_elements) { // Ensure each thread processes a valid element
        float t = output[idx];
        pre_grad[idx] *= t * (1 - t); // Compute the gradient in a more memory-coalesced manner
    }
}