#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
Modified from
https://github.com/zhxfl/CUDA-CNN
*/

__global__ void elementwiseMul(float *x, float *y, float *z, int rows, int cols) {
    int globalIdx = blockIdx.y * blockDim.y * cols + blockIdx.x * blockDim.x + threadIdx.y * cols + threadIdx.x;

    if (globalIdx < rows * cols) {
        // Perform element-wise multiplication
        z[globalIdx] = x[globalIdx] * y[globalIdx];
    }
}