#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bias_grad(float *pre_grad, float *output, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < rows) {
        float sum = 0.0f; // Use a local variable to store the sum
        for (int k = 0; k < cols; k++) {
            sum += pre_grad[i * cols + k];
        }
        output[i] = sum; // Write the accumulated sum to the output
    }
}