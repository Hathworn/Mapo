#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized ReLU gradient kernel
__global__ void relu_grad(float *pre_grad, float *output, int rows, int cols) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Check within bounds and output value simultaneously
    if (i < rows && j < cols && output[i * cols + j] <= 0) {
        pre_grad[i * cols + j] = 0;
    }
}