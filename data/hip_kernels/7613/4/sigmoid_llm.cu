#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sigmoid(float *inout, float *bias, int rows, int cols) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds only once
    if (j < cols && i < rows) {
        // Calculate index once
        int index = i * cols + j;
        float t = inout[index];
        // Reduce expf call and reuse computed index
        inout[index] = 1 / (1 + expf(-t)) + bias[i];
    }
}