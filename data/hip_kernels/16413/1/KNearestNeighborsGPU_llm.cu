#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

void KNearestNeighborsCPU(float3 *dataArray, int *result, int cnt);

// cpu algorithm
__global__ void KNearestNeighborsGPU(float3 *dataArray, int *result, int cnt)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= cnt) return;

    float3 point = dataArray[id];
    float minimumDist = FLT_MAX; // Use pre-defined maximum value
    float3 current;
    float distance;

    for (int j = 0; j < cnt; ++j)
    {
        if (id == j) continue;

        current = dataArray[j];
        // Calculate distance using common subexpression elimination
        float dx = point.x - current.x;
        float dy = point.y - current.y;
        float dz = point.z - current.z;

        distance = dx * dx + dy * dy + dz * dz;
        
        if (distance < minimumDist)
        {
            minimumDist = distance;
            result[id] = j;
        }
    }
}