#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binZeros(int *d_bin_count, int bin_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use loop unrolling for better performance if the bin_size is a known multiple
    if (i < bin_size) {
        d_bin_count[i] = 0;
        if (i + blockDim.x < bin_size) d_bin_count[i + blockDim.x] = 0;
        if (i + 2 * blockDim.x < bin_size) d_bin_count[i + 2 * blockDim.x] = 0;
        if (i + 3 * blockDim.x < bin_size) d_bin_count[i + 3 * blockDim.x] = 0;
    }
}