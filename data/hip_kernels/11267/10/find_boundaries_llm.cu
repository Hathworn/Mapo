#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void find_boundaries(const int num_keys, const int num_bucket, const int *which_bucket, int *bucket_start) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation

    if (index < num_keys) { // Ensure the thread index is within range
        int previous_bucket = (index > 0 ? which_bucket[index - 1] : -1);
        int my_bucket = which_bucket[index];

        // Check for boundary change
        if (previous_bucket != my_bucket) {
            bucket_start[my_bucket] = index; // Set the start of the new bucket
        }

        // Handle the last element separately
        if (index == num_keys - 1) {
            for (int i = my_bucket + 1; i < num_bucket; ++i) {
                bucket_start[i] = num_keys; // Set remaining buckets to num_keys
            }
        }
    }
}