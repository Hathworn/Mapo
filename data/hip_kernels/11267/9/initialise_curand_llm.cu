#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel to reduce unnecessary synchronization and improve parallel execution
__global__ void initialise_curand(hiprandState * state, unsigned long seed)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique thread index
    hiprand_init(seed, idx, 0, &state[idx]); // Initialize the PRNG state without synchronization
}