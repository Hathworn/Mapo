#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initialise_curand(hiprandState * state, unsigned long seed)
{
int idx = blockIdx.x*blockDim.x + threadIdx.x;
hiprand_init(seed, idx, 0, &state[idx]);
//printf("index idx = %d", idx);
__syncthreads();
}