#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ Query query_table(const int num_bucket, const int *bucket_start, const int key) {
    const unsigned int bucket_id = key;
    const unsigned int list_start = (bucket_id > 0 ? bucket_start[bucket_id - 1] : 0);
    const unsigned int next_list_start = bucket_start[bucket_id];
    Query query(list_start, next_list_start);
    return query;
}

__global__ void queryDevice(const int num_bucket, const int *bucket_start, const int key) {
    // Optimize by using fewer registers and improve warp efficiency
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_bucket) {
        Query queryresult = query_table(num_bucket, bucket_start, key);
    }
}