#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Use block and grid index for processing larger arrays
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // No bounds checking assuming the caller ensures adequate grid/block sizes
    c[i] = a[i] + b[i];
}