#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int local_factor = *factor; // Cache factor in a shared register

    // Unroll loop for better performance
    #pragma unroll
    for (int i = 0; i < num_iterations; i++)
    {
        g_data[idx] += local_factor;
    }
}