#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void updateVelocity_k(float2 *v, float *vx, float *vy, int dx, int pdx, int dy, int lb, size_t pitch) {
    // Calculate unique global thread ID
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy_base = blockIdx.y * (lb * blockDim.y);
    
    // Precompute scale factor outside loop
    float scale = 1.f / (dx * dy);

    // Check boundary condition for x domain
    if (gtidx < dx) {
        for (int p = 0; p < lb; ++p) {
            int gtidy = gtidy_base + threadIdx.y * lb + p;
            
            // Check boundary condition for y domain
            if (gtidy < dy) {
                int fjr = gtidy * pdx + gtidx;
                
                // Load vxterm and vyterm, Normalize and store result
                float vxterm = vx[fjr] * scale;
                float vyterm = vy[fjr] * scale;
                
                // Directly assign normalized result to global memory
                ((float2*)((char*)v + gtidy * pitch))[gtidx] = make_float2(vxterm, vyterm);
            }
        }
    }
}