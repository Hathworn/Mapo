#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(int *array, int arrayCount)
{
    // Calculate global thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Check if within bounds and perform squaring
    if (idx < arrayCount) {
        array[idx] *= array[idx];
    }
}