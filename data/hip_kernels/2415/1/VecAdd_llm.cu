#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAdd(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int N)
{
    // Calculate the global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Process multiple elements per thread to reduce global memory transactions
    int stride = gridDim.x * blockDim.x;
    
    for (int index = i; index < N; index += stride) {
        C[index] = A[index] + B[index];
    }
}