#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addForces_k(float2 *v, int dx, int dy, int spx, int spy, float fx, float fy, int r, size_t pitch) {

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Pre-compute often used values
    int tx_r = tx - r;
    int ty_r = ty - r;
    float tx_r4 = tx_r * tx_r * tx_r * tx_r;
    float ty_r4 = ty_r * ty_r * ty_r * ty_r;
    
    // Get pointer to target element in pitched memory
    float2 *fj = (float2*)((char*)v + (ty + spy) * pitch) + tx + spx;
    
    // Fetch current force value
    float2 vterm = *fj;
    
    // Compute scaling factor once
    float s = 1.f / (1.f + tx_r4 + ty_r4);
    
    // Update forces
    vterm.x += s * fx;
    vterm.y += s * fy;
    
    // Write back updated force
    *fj = vterm;
}