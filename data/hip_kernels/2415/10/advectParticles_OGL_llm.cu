#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void advectParticles_OGL(float2 *part, float2 *v, int dx, int dy, float dt, int lb, size_t pitch) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    if (gtidx < dx) {
        for (p = 0; p < lb; p++) {
            int fi = gtidy + p;
            if (fi < dy) {
                int fj = fi * dx + gtidx;
                float2 pterm = part[fj];

                int xvi = __float2int_rd(pterm.x * dx); // Use faster float to int conversion
                int yvi = __float2int_rd(pterm.y * dy);

                float2 vterm = *((float2*)((char*)v + yvi * pitch) + xvi);

                // Use fmodf to simplify periodic boundary conditions
                pterm.x = fmodf(pterm.x + dt * vterm.x + 1.f, 1.f);
                pterm.y = fmodf(pterm.y + dt * vterm.y + 1.f, 1.f);

                part[fj] = pterm;
            }
        }
    }
}