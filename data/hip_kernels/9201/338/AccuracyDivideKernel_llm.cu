#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use local variable to reduce global memory accesses
    float acc = *accuracy;
    acc /= N;
    *accuracy = acc;
}