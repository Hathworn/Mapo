#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef DTYPE
#define DTYPE float
#endif

}

__global__ void tensor_1d_equals(const int n, const DTYPE* __restrict__ x, const int offset_x, const int stride_x, const DTYPE* __restrict__ y, const int offset_y, const int stride_y, int* __restrict__ eq_flag) {
    // Calculate global thread ID
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread is within bounds
    if (gid < n) {
        const int ix = offset_x + gid * stride_x;
        const int iy = offset_y + gid * stride_y;
        
        // Perform comparison and increment if not equal
        if (x[ix] != y[iy]) {
            atomicAdd(eq_flag, 1); // Use atomic to avoid race conditions
        }
    }
}