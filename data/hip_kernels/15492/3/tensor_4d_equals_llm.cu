#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef DTYPE
#define DTYPE float
#endif

}

__global__ void tensor_4d_equals(const int n, const int c, const int h, const int w, const DTYPE* x, const int offset_x, const int n_x, const int c_x, const int h_x, const int w_x, const DTYPE* y, const int offset_y, const int n_y, const int c_y, const int h_y, const int w_y, int* eq_flag) {
    const int gid_n = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_c = blockIdx.y * blockDim.y + threadIdx.y;
    const int gid_h = blockIdx.z * blockDim.z + threadIdx.z;
    const bool valid = (gid_n < n) && (gid_c < c) && (gid_h < h);

    // Use shared memory to reduce memory transaction overhead
    __shared__ int local_eq_flag;
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
        local_eq_flag = 0;
    }
    __syncthreads();

    if (valid) {
        const int ix = offset_x + gid_n * n_x + gid_c * c_x + gid_h * h_x;
        const int iy = offset_y + gid_n * n_y + gid_c * c_y + gid_h * h_y;
        for (int i = 0; i < w; i++) {
            if (x[ix + i * w_x] != y[iy + i * w_y]) {
                atomicAdd(&local_eq_flag, 1);  // Use atomic operation to update flag
            }
        }
    }
    
    __syncthreads();
    
    // Conditionally move from shared to global memory to avoid race
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
        atomicAdd(eq_flag, local_eq_flag);
    }
}