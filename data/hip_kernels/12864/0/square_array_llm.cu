#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Using a more informative kernel name and simplifying the logic
__global__ void squareArrayOptimized(float *a, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) 
    {
        float val = a[idx]; // Load once
        a[idx] = val * val; // Compute square
    }
}