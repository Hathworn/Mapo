#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory to reduce global memory access
    __shared__ float shared_accuracy;
    
    // Load the data into shared memory
    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy;
    }

    // Ensure all threads wait until shared memory is loaded
    __syncthreads();

    // Perform the division in only one thread
    if (threadIdx.x == 0) {
        shared_accuracy /= N;
        *accuracy = shared_accuracy;
    }
}