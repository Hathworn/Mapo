#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform bounds check to ensure threads don't access out-of-bounds memory
    if (i < N)    
    {
        // Perform addition
        c[i] = a[i] + b[i];
    }
}