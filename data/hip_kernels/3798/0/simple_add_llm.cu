#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_add(float* __restrict__ a, const float* __restrict__ b, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Process in a loop to potentially enhance performance for very large arrays
    // by utilizing thread block size more efficiently
    for(; i < n; i += blockDim.x * gridDim.x) {
        a[i] += b[i];
    }
}