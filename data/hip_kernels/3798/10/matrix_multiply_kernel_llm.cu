#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    // Load the thread index and calculate the row index for this thread
    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < dim_mn) {
        double out = 0.0;

        // Use shared memory to cache the vector_in for all threads in the block
        extern __shared__ double shared_vector_in[];

        if (threadIdx.x < dim_mn) {
            shared_vector_in[threadIdx.x] = vector_in[threadIdx.x];
        }
        __syncthreads();

        for (long j = 0; j < dim_mn; j++) {
            // Access shared memory instead of global memory for vector_in
            out += matrix[i * dim_mn + j] * shared_vector_in[j];
        }

        vector_out[i] = out;
    }
}