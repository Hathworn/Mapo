#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction(int* input, int* output) {
    __shared__ int tmp[TPB];

    // Load input elements into shared memory
    int tid = threadIdx.x;
    int idx = tid + blockIdx.x * blockDim.x;
    tmp[tid] = input[idx];
    
    __syncthreads();

    // Perform reduction using a single loop
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            tmp[tid] += tmp[tid + s];
        }
        __syncthreads();
    }

    // Write the result from the first thread of each block
    if (tid == 0) {
        output[blockIdx.x] = tmp[0];
    }
}