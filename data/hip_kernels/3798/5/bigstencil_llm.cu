#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bigstencil(int* __restrict__ in, int* __restrict__ out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Load data from global memory to register
    int temp = in[i];
    // Perform computation
    temp += 2;
    // Store results back to global memory
    out[i] = temp;
}