#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void modcpy(void *destination, void *source, size_t destination_size, size_t source_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Cache grid stride for efficiency

    int ds = destination_size / sizeof(int4);
    int ss = source_size / sizeof(int4);

    // Use unrolled loop for better performance
    for (int i = idx; i < ds; i += stride) {
        int pos = i % ss;
        reinterpret_cast<int4*>(destination)[i] = reinterpret_cast<int4*>(source)[pos];
    }
}