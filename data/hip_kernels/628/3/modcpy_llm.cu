#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void modcpy(void *destination, void *source, size_t destination_size, size_t source_size) {
    // Calculate thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Divide sizes by sizeof(int4) once to avoid redundant computations
    int ds = destination_size / sizeof(int4);
    int ss = source_size / sizeof(int4);

    // Optimize loop by eliminating division inside the loop
    for (int i = idx; i < ds; i += gridDim.x * blockDim.x) {
        int pos = i % ss; // Compute source position once per iteration
        reinterpret_cast<int4*>(destination)[i] = reinterpret_cast<int4*>(source)[pos]; // Copy data
    }
}