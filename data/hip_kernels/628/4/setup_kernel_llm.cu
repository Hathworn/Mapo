#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(long long seed, hiprandState_t* devStates, ushort N) {
    // Calculate global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize loop by calculating stride once
    int stride = gridDim.x * blockDim.x;

    // Loop over all necessary indices using calculated stride
    for (int i = idx; i < N; i += stride) {
        hiprand_init(seed, i, 0, &devStates[i]);
    }
}