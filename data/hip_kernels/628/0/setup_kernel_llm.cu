#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(long long seed, hiprandState_t* devStates, ushort N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Compute the stride for the loop
    for (int i = idx; i < N; i += stride) { // Use precomputed stride in loop
        hiprand_init(seed, i, 0, &devStates[i]);
    }
}