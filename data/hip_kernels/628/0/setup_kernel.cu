#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_kernel(long long seed, hiprandState_t* devStates, ushort N){
int idx = blockIdx.x * blockDim.x + threadIdx.x;
int i;
for(i=idx;i<N;i+=gridDim.x*blockDim.x)
hiprand_init(seed, i, 0, &devStates[i]);
}