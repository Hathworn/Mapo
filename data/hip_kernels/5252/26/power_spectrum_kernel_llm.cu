#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo) {
    // Use shared memory for optimization if needed.
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;

    // Loop unrolling for better performance
    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {
        if (idx == 0) continue;

        // Use float2 loads for better efficiency
        float2 val = reinterpret_cast<float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = ret;
    }
    
    // Handle special case in a separate, optimized branch
    if (thread_id == 0) {
        float real = Ar[0];
        float im = Ar[row_length];
        Aw[0] = real * real;
        Aw[half_length] = im * im;
    }
}