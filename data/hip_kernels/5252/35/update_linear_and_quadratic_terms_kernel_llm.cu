#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel(int32_t n, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    float val = 1.0f;
    float cur_weight = *cur_tot_weight;

    // Pre-calculate new_scale and val, and avoid recomputation in the loop
    if (max_count > 0.0f) {
        float new_scale = max(cur_weight, (float)max_count) / max_count;
        val += new_scale - 1.0f;
    }

    // Utilize variables for indexing to avoid repeated calculation
    int32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride = blockDim.x * gridDim.x;

    for (int32_t i = tid; i < n; i += stride) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        quadratic[diag_idx] += val;
    }

    // Use one thread per block for a reduction on linear, to avoid collisions
    if (tid == 0) {
        atomicAdd(&linear[0], val * prior_offset);
    }
}