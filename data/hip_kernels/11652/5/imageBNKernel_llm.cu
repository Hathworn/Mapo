#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void imageBNKernel(unsigned char* d_image, int h, int w)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if Row and Col are within image bounds
    if ((Row < h) && (Col < w)) {
        int pixelIndex = 4 * (w * Row + Col); // Calculate pixel index once

        int r = d_image[pixelIndex + 0];
        int g = d_image[pixelIndex + 1];
        int b = d_image[pixelIndex + 2];

        d_image[pixelIndex + 0] = 0; // Set R channel to 0
        d_image[pixelIndex + 1] = 0; // Set G channel to 0
        d_image[pixelIndex + 2] = 0; // Set B channel to 0

        // Convert to grayscale and assign to A channel
        d_image[pixelIndex + 3] = static_cast<unsigned char>(r * 0.21f + g * 0.71f + b * 0.07f);
    }
}