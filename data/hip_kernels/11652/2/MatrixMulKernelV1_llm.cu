#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulKernelV1(float* M, float* N, float* P, int Width)
{
    // Calculate row and column based on thread/block position
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if (Row < Width && Col < Width) {
        float Pvalue = 0.0f;  // Initialize partial result

        // Loop unrolling for performance improvement
        for (int k = 0; k < Width; k += 4) {
            Pvalue += M[Row * Width + k] * N[k * Width + Col];
            if (k + 1 < Width) Pvalue += M[Row * Width + k + 1] * N[(k + 1) * Width + Col];
            if (k + 2 < Width) Pvalue += M[Row * Width + k + 2] * N[(k + 2) * Width + Col];
            if (k + 3 < Width) Pvalue += M[Row * Width + k + 3] * N[(k + 3) * Width + Col];
        }

        // Write result to memory
        P[Row * Width + Col] = Pvalue;
    }
}