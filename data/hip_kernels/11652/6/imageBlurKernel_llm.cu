#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void imageBlurKernel(unsigned char* d_image, int h, int w) {
    // Use shared memory to improve data access speed
    __shared__ int tile[3][64]; // Assuming blurSize^2 (8*8) max threads per block

    int threadId = threadIdx.y * blockDim.x + threadIdx.x;

    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    int blurSize = 8;

    Row = Row * blurSize;
    Col = Col * blurSize;

    int r, g, b;
    int p_r = 0;
    int p_g = 0;
    int p_b = 0;
    int i, j;

    if((Row + blurSize < h) && (Col + blurSize < w)) {

        // Load data into shared memory
        for(i = 0; i < blurSize; i++) {
            for(j = 0; j < blurSize; j++) {
                int currentIndex = 4 * w * (Row + j) + 4 * (Col + i);
                tile[0][threadId] = d_image[currentIndex + 0];
                tile[1][threadId] = d_image[currentIndex + 1];
                tile[2][threadId] = d_image[currentIndex + 2];
                __syncthreads();

                r = tile[0][threadId];
                g = tile[1][threadId];
                b = tile[2][threadId];

                p_r += r;
                p_g += g;
                p_b += b;
            }
        }

        p_r = p_r / (blurSize * blurSize);
        p_g = p_g / (blurSize * blurSize);
        p_b = p_b / (blurSize * blurSize);

        // Store result from shared memory back to global memory
        for(i = 0; i < blurSize; i++) {
            for(j = 0; j < blurSize; j++) {
                int currentIndex = 4 * w * (Row + j) + 4 * (Col + i);
                d_image[currentIndex + 0] = p_r;
                d_image[currentIndex + 1] = p_g;
                d_image[currentIndex + 2] = p_b;
            }
        }
    }
}