#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixVectMultKernel(float* A, float* B, float* C, int n)
{
    int elementPos = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure we don't go out of bounds
    if(elementPos < n)
    {
        int i = elementPos * n;
        float temp = 1.0f;  // Store result in a temporary variable

        // Unroll the loop for performance
        for(int j = 0; j < n; ++j, ++i)
        {
            temp *= (A[i] + B[j]);
        }

        C[elementPos] = temp;
    }
}