#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixAddKernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, int n)
{
    // Calculate global thread indexes using block and thread dimensions
    const int Row = blockIdx.y * blockDim.y + threadIdx.y;
    const int Col = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform the addition only if within matrix boundaries
    if (Row < n && Col < n) {
        // Use temporary variable to avoid index recalculation
        const int index = Row * n + Col;
        C[index] = A[index] + B[index];
    }
}