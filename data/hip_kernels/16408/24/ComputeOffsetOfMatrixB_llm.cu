#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Calculate global thread ID
    int32_t global_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride pattern for efficiency
    for (int32_t i = global_id; i < N * gridDim.x; i += blockDim.x * gridDim.x) {
        int32_t row_index = i / N;
        int32_t col_index = i % N;
        output[i] = -row_sum[row_index];
    }
}