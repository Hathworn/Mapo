#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Calculate the global thread index.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Utilize shared memory for storing batch and class index to reduce global memory access.
    __shared__ int64_t s_batch_index;
    __shared__ int64_t s_class_index;

    if (threadIdx.x == 0) {
        s_batch_index = batch_index;
        s_class_index = class_index;
    }
    __syncthreads();

    // Iterate over elements with stride and use shared memory indexes.
    for (; idx < num_elements; idx += blockDim.x * gridDim.x) {
        int base_idx = idx * 3;
        to_normalize[base_idx]     = s_batch_index;
        to_normalize[base_idx + 1] = s_class_index;
        to_normalize[base_idx + 2] = static_cast<int64_t>(original[idx]);
    }
}