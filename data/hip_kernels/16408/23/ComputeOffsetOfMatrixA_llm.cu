#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    // Calculate global thread index
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Process elements with strides of total number of threads
    for (int32_t i = idx; i < N; i += blockDim.x * gridDim.x) {
        output[blockIdx.x * N + i] = -col_sum[i];
    }
}