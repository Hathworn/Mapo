#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixAB(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, int32_t K_A_B, int32_t N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Compute global thread index
    if (i < N) {  // Only compute if within bounds
        output[blockIdx.x * N + i] = K_A_B - row_sum[blockIdx.x] - col_sum[i];
    }
}