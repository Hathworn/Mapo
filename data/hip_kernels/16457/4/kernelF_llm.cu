#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelF(const float *d_xAx, const float *d_bx, const float *d_c, float *d_y)
{
    // Use block and thread indices for parallel computation
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check array bounds; assume d_y is pre-allocated with the correct size
    if (idx < 1)  // Assuming we only need 1 element; otherwise adjust according to array size
    {
        d_y[idx] = d_xAx[idx] + d_bx[idx] + d_c[idx];
    }
}