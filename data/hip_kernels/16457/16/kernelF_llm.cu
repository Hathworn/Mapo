#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelF(const float *d_x, float *d_y)
{
    // Load first two elements from global memory and store in registers
    float x0 = d_x[0];
    float x1 = d_x[1];

    // Precompute common subexpressions
    float x0_square = x0 * x0;
    float a = (1.0f - x0);
    float b = (x1 - x0_square);

    // Compute result and store in output
    *d_y = (a * a) + 100.0f * (b * b);
}