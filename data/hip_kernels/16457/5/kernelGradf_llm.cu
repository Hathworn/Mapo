#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelGradf(const float *d_x, float *d_grad, const float *A, const float *b, const size_t len)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index >= len)
        return;

    float grad = 0.0f; // Use local variable to reduce global memory access

    for (size_t j = 0; j < len; ++j)
    {
        grad += A[index * len + j] * d_x[j];
    }

    d_grad[index] = grad * 2.0f + b[index]; // Combine operations to minimize write operations
}