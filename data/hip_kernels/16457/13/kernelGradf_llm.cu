#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelGradf(const float *d_x, float *d_grad, float *A, float *b, const size_t len)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= len)
        return;

    float temp = 0.0f; // Initialize temporary variable

    // Use shared memory for improved access speed (optimize memory access pattern)
    __shared__ float shared_d_x[1024]; // Assuming blockDim.x <= 1024
    for (size_t j = threadIdx.x; j < len; j += blockDim.x)
    {
        shared_d_x[j] = d_x[j];
    }
    __syncthreads(); // Synchronize threads to ensure shared memory is populated

    for (size_t j = 0; j < len; ++j)
    {
        temp += A[index * len + j] * shared_d_x[j];
    }

    d_grad[index] = temp * 2.0f + b[index]; // Combine operations for better optimization
}