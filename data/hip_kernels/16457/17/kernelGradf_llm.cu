#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelGradf(const float *d_x, float *d_grad)
{
    // Utilize registers for intermediate computation
    float x0 = d_x[0];
    float x1 = d_x[1];
    float x0_squared = x0 * x0;
    float common_term = x1 - x0_squared;

    // Optimize calculations by reusing computed values
    d_grad[0] = -2.0f * (1.0f - x0) - 400.0f * x0 * common_term;
    d_grad[1] = 200.0f * common_term;
}