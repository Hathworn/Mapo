#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelF(const float *d_xAx, const float *d_bx, const float *d_c, float *d_y)
{
    // Calculate thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check index bounds
    if (idx < 1) {
        d_y[idx] = d_xAx[idx] + d_bx[idx] + d_c[idx];
    }
}