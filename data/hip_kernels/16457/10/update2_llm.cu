#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update2(float *alphaMinusBeta_out, const float *rho, const float *yDotZ, const float *alpha)
{
    // Avoid repeated memory access by using local variables
    float local_rho = *rho;
    float local_yDotZ = *yDotZ;
    float local_alpha = *alpha;
    float beta = local_rho * local_yDotZ;
    *alphaMinusBeta_out = local_alpha - beta;
}