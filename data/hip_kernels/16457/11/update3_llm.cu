#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update3(float *rho_out, float *H0_out, const float *yDotS, const float *yDotY)
{
    // Load yDotS and yDotY into registers for faster access
    float yDotS_val = *yDotS;
    float yDotY_val = *yDotY;

    // Update rho_out and H0_out with conditional check inside
    *rho_out = 1.0f / yDotS_val;
    if (yDotY_val > 1e-5)
    {
        *H0_out = yDotS_val / yDotY_val;
    }
}