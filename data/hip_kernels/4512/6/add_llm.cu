#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int n, float *a, float *b, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for the loop
    
    // Use stride loop for improved parallel efficiency
    for (; i < n; i += stride)
    {
        sum[i] = a[i] + b[i];
    }
}