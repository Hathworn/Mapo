#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define CONSTANT_MEMORY_SIZE 100
__constant__ float constantMemoryData[CONSTANT_MEMORY_SIZE];

extern "C"
__global__ void constantMemoryKernel(float* array, int size)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (index < size && index < CONSTANT_MEMORY_SIZE) {
        // Copy constant memory data to global memory
        array[index] = constantMemoryData[index];
    }
}