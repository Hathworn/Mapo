#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void childKernel(unsigned int parentThreadIndex, float* data)
{
    // Using a register variable for thread index to avoid multiple memory accesses
    unsigned int tid = threadIdx.x;
    
    // Minimize printf calls for better performance
    if (tid == 0) {
        printf("Parent thread index: %d\n", parentThreadIndex);
    }
    
    // Perform computation and data storage
    data[tid] = parentThreadIndex + 0.1f * tid;
}