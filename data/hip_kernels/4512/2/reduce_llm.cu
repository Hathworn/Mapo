#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void reduce(float *g_idata, float *g_odata, unsigned int n)
{
    extern __shared__ float sdata[];

    // First level reduction: global to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x*2 + tid;
    unsigned int gridSize = blockDim.x*2*gridDim.x;

    float mySum = 0;

    // Unroll loop for better performance
    while (i < n) {
        mySum += g_idata[i];
        if (i + blockDim.x < n) mySum += g_idata[i+blockDim.x];
        i += gridSize;
    }

    // Store local sum in shared memory
    sdata[tid] = mySum;
    __syncthreads();

    // Reduce in shared memory
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Warp reduction (no synchronization needed within a warp)
    if (tid < 32) {
        volatile float* smem = sdata;  // Declare as volatile
        smem[tid] += smem[tid + 32];
        smem[tid] += smem[tid + 16];
        smem[tid] += smem[tid + 8];
        smem[tid] += smem[tid + 4];
        smem[tid] += smem[tid + 2];
        smem[tid] += smem[tid + 1];
    }

    // Write result to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}