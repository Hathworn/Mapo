#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups() {
    // Use __threadfence() to ensure memory operations are completed before synchronization
    __threadfence();

    // Use __syncthreads() for synchronization within the block
    __syncthreads();
}