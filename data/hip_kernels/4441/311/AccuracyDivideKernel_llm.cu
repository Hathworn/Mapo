#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel by using thread synchronization for shared reduction
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    __shared__ float shared_acc;
    if (threadIdx.x == 0) {
        shared_acc = *accuracy; // Load accuracy into shared memory
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        shared_acc /= N; // Perform division from shared memory
        *accuracy = shared_acc; // Write result back to global memory
    }
}