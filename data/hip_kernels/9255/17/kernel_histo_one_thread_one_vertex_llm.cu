#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_histo_one_thread_one_vertex( unsigned int *ct, unsigned int *histo ){

    // Unique thread id
    unsigned int tid_x = threadIdx.x + blockDim.x * blockIdx.x;

    // Calculate start index for each vertex
    unsigned int start_idx = tid_x * constant_n_hits;

    // Loop for each hit in the vertex
    for (unsigned int ihit = 0; ihit < constant_n_hits; ihit++) {
        unsigned int bin = start_idx + ihit;
        if (bin < constant_n_test_vertices * constant_n_hits) {
            // Atomic update to histogram
            atomicAdd(&histo[ct[bin]], 1);
        }
    }
}