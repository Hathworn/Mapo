#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_histo_iterated(unsigned int *ct, unsigned int *histo, unsigned int offset) {
    extern __shared__ unsigned int temp[];
    unsigned int index = threadIdx.x;
    temp[index] = 0;
    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int size = blockDim.x * gridDim.x;
    unsigned int max = constant_n_hits * constant_n_test_vertices;

    // Loop through data in chunks
    while (i < max) {
        atomicAdd(&temp[ct[i]], 1);
        i += size;
    }

    __syncthreads();

    // Reduce shared memory accumulation to global memory
    atomicAdd(&(histo[index + offset]), temp[index]);
}