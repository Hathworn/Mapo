#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_histo_per_vertex_shared(unsigned int *ct, unsigned int *histo) {
    // Get unique id for each thread in each block
    unsigned int tid_x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid_y = threadIdx.y + blockDim.y * blockIdx.y;

    if (tid_x >= constant_n_test_vertices) return;

    unsigned int vertex_offset = tid_x * constant_n_hits;
    unsigned int stride_block = blockDim.y;
    unsigned int ihit = vertex_offset + tid_y;
    unsigned int time_offset = tid_x * constant_n_time_bins;

    unsigned int local_ihit = threadIdx.y;

    // Use shared memory dynamically allocated
    extern __shared__ unsigned int temp[];

    // Simplified initialization of shared memory
    for (unsigned int i = local_ihit; i < constant_n_time_bins; i += stride_block) {
        temp[i] = 0;
    }
    
    __syncthreads();

    // Process hits with the step size across threads
    unsigned int stride = blockDim.y * gridDim.y;
    for (; ihit < vertex_offset + constant_n_hits; ihit += stride) {
        unsigned int bin = ct[ihit];
        atomicAdd(&temp[bin - time_offset], 1);
    }

    __syncthreads();

    // Atomic addition of results from shared memory to global memory
    local_ihit = threadIdx.y;
    for (unsigned int i = local_ihit; i < constant_n_time_bins; i += stride_block) {
        atomicAdd(&histo[i + time_offset], temp[i]);
    }
}