#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_histo_stride(unsigned int *ct, unsigned int *histo) {
    // Calculate the global thread index
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory to reduce contention on histo
    extern __shared__ unsigned int shared_histo[];
    int tid = threadIdx.x;
    
    // Initialize shared memory histogram
    for (int j = tid; j < constant_bin_count; j += blockDim.x) {
        shared_histo[j] = 0;
    }
    __syncthreads();

    // Accumulate histogram in shared memory
    while (i < constant_n_hits * constant_n_test_vertices) {
        atomicAdd(&shared_histo[ct[i]], 1);
        i += stride;
    }
    __syncthreads();

    // Reduce shared memory histogram to global memory
    for (int j = tid; j < constant_bin_count; j += blockDim.x) {
        atomicAdd(&histo[j], shared_histo[j]);
    }
}