#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_histo_iterated(unsigned int *ct, unsigned int *histo, unsigned int offset) {

    extern __shared__ unsigned int temp[];
    unsigned int index = threadIdx.x + offset;
    temp[threadIdx.x] = 0;  // Initialize shared memory using thread index for optimal performance
    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int size = blockDim.x * gridDim.x;
    unsigned int max = constant_n_hits * constant_n_test_vertices;

    while (i < max) {
        atomicAdd(&temp[ct[i] - offset], 1);  // Adjusted for shared memory index
        i += size;
    }
    __syncthreads();

    // Efficient atomic operation, one per block thread
    unsigned int histoIndex = index - offset;  // Calculate histogram index offset
    if (threadIdx.x == 0) {  // One thread aggregates all data in shared memory
        for (int j = 0; j < blockDim.x; ++j) {
            atomicAdd(&(histo[histoIndex + j]), temp[j]);
        }
    }
}