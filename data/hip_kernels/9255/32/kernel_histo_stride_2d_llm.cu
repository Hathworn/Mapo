#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_histo_stride_2d(unsigned int *ct, unsigned int *histo) {
    // Calculate unique thread ID for 2D grid
    unsigned int tid_x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid_y = threadIdx.y + blockDim.y * blockIdx.y;
    unsigned int size = blockDim.x * gridDim.x;
    unsigned int max = constant_n_hits * constant_n_test_vertices;

    // Flatten 2D indices to 1D index
    int tid = tid_y * size + tid_x;
    unsigned int stride = blockDim.y * gridDim.y * size;

    // Loop over data with thread-level stride
    while (tid < max) {
        atomicAdd(&histo[ct[tid]], 1);  // Perform atomic addition
        tid += stride;  // Advance thread index by stride
    }
}