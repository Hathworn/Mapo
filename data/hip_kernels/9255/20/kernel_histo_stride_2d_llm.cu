#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_histo_stride_2d(unsigned int *ct, unsigned int *histo) {

    // Get unique id for each thread in each block
    unsigned int tid_x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid_y = threadIdx.y + blockDim.y * blockIdx.y;
    
    unsigned int size = blockDim.x * gridDim.x;
    unsigned int max = constant_n_hits * constant_n_test_vertices;
    
    // Map the two 2D indices to a single linear, 1D index
    int tid = tid_y * size + tid_x;
    
    // Calculate stride using both dimensions' sizes
    unsigned int stride = blockDim.y * gridDim.y * size;
    
    // Iterate over data with efficient stride loop
    while (tid < max) {
        atomicAdd(&histo[ct[tid]], 1);
        tid += stride;
    }
}