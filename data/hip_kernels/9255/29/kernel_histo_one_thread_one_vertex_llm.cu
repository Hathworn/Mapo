#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_histo_one_thread_one_vertex( unsigned int *ct, unsigned int *histo ){

// Get unique id for each thread in each block
unsigned int tid_x = threadIdx.x + blockDim.x * blockIdx.x;

// Precompute values outside the loop
unsigned int vertex_index = tid_x;
unsigned int bin, max = constant_n_test_vertices * constant_n_hits;
unsigned int size = vertex_index * constant_n_hits;

#pragma unroll // Use loop unrolling for better performance
for( unsigned int ihit = 0; ihit < constant_n_hits; ihit++){
    bin = size + ihit;
    if( bin < max) {
        atomicAdd(&histo[ct[bin]], 1);
    }
}

}