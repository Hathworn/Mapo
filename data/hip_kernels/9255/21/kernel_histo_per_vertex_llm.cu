#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_histo_per_vertex( unsigned int *ct, unsigned int *histo) {

    // Calculate unique id for each thread in the grid
    unsigned int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check if thread is within bounds
    if (tid_x >= constant_n_test_vertices) return;

    unsigned int vertex_offset = tid_x * constant_n_hits;
    unsigned int bin;
    unsigned int stride = blockDim.y * gridDim.y;
    unsigned int ihit = vertex_offset + tid_y;

    // Loop through hits with a stride to access memory efficiently
    while (ihit < vertex_offset + constant_n_hits) {
        bin = ct[ihit];
        atomicAdd(&histo[bin], 1);  // Atomic add to avoid race conditions
        ihit += stride;
    }
}