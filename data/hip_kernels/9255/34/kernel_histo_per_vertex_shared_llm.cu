#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_histo_per_vertex_shared(unsigned int *ct, unsigned int *histo) {
    unsigned int tid_x = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid_y = threadIdx.y;
    if (tid_x >= constant_n_test_vertices) return;

    unsigned int vertex_offset = tid_x * constant_n_hits;
    unsigned int bin;
    unsigned int stride_block = blockDim.y;
    unsigned int ihit = vertex_offset + tid_y;
    unsigned int time_offset = tid_x * constant_n_time_bins;

    extern __shared__ unsigned int temp[];
    
    // Initialize shared memory
    for (unsigned int i = tid_y; i < constant_n_time_bins; i += stride_block) {
        temp[i] = 0;
    }
    __syncthreads();

    // Compute histogram in shared memory
    for (unsigned int i = ihit; i < vertex_offset + constant_n_hits; i += stride_block) {
        bin = ct[i];
        atomicAdd(&temp[bin - time_offset], 1);
    }
    __syncthreads();

    // Write results from shared memory to global memory
    for (unsigned int i = tid_y; i < constant_n_time_bins; i += stride_block) {
        atomicAdd(&histo[i + time_offset], temp[i]);
    }
}