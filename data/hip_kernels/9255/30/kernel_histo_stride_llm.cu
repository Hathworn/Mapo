#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_histo_stride(unsigned int *ct, unsigned int *histo) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop unrolling for improved performance
    for (int i = index; i < constant_n_hits * constant_n_test_vertices; i += stride * 4) {
        atomicAdd(&histo[ct[i]], 1);
        if (i + stride < constant_n_hits * constant_n_test_vertices) 
            atomicAdd(&histo[ct[i + stride]], 1);
        if (i + 2 * stride < constant_n_hits * constant_n_test_vertices) 
            atomicAdd(&histo[ct[i + 2 * stride]], 1);
        if (i + 3 * stride < constant_n_hits * constant_n_test_vertices) 
            atomicAdd(&histo[ct[i + 3 * stride]], 1);
    }
}