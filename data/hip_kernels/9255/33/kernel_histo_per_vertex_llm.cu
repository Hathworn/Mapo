#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_histo_per_vertex(unsigned int *ct, unsigned int *histo) {
    // Calculate global thread ID
    unsigned int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tid_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (tid_x >= constant_n_test_vertices) return;

    unsigned int vertex_offset = tid_x * constant_n_hits;
    unsigned int stride = blockDim.y * gridDim.y;
    unsigned int ihit = vertex_offset + tid_y;

    // Loop over hits with strided access
    while (ihit < vertex_offset + constant_n_hits) {
        unsigned int bin = ct[ihit];
        atomicAdd(&histo[bin], 1);
        ihit += stride;
    }

    // Ensure all threads reach this point
    __syncthreads();
}
```
