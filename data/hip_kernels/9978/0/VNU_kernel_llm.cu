#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifndef _VNU_KERNEL_H_
#define _VNU_KERNEL_H_

#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#define database_character( index)  CUT_BANK_CHECKER(database_character, index)
#define temp_1( index)              CUT_BANK_CHECKER(temp_1,             index)
#define temp_2( index)              CUT_BANK_CHECKER(temp_2,             index)

#endif // #ifndef _VNU_KERNEL_H_

#ifndef _CNU_KERNEL_H_
#define _CNU_KERNEL_H_

#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#define database_character( index)  CUT_BANK_CHECKER(database_character, index)
#define temp_1( index)              CUT_BANK_CHECKER(temp_1,             index)
#define temp_2( index)              CUT_BANK_CHECKER(temp_2,             index)

#endif // #ifndef _CNU_KERNEL_H_

__global__ void VNU_kernel(short int* device_array, short int* offset_array, short int* sign_array, short int* results_array) {
    extern __shared__ short int shared[]; // Use external shared memory
    short int* offset = shared; 
    short int* current_Index = &shared[blockDim.x];

    int thread_Id = threadIdx.x;
    *current_Index = ((blockIdx.x * blockDim.x + thread_Id) * 2); // Calculate current_Index

    // Load offset into shared memory
    offset[thread_Id] = offset_array[*current_Index];
    __syncthreads(); // Ensure all offsets are loaded before computation

    short int input1 = results_array[*current_Index + offset[thread_Id]];
    short int input2 = results_array[*current_Index + offset[thread_Id] + 1];
    short int input3 = device_array[(*current_Index / 2) + offset[thread_Id]];

    short int sum = (input1 + input2 + input3);
    short int output1 = (sum - input1);
    short int output2 = (sum - input2);
    short int sign = (sum < 0) ? 1 : 0; // Condensed sign determination

    results_array[*current_Index + offset[thread_Id]] = output1;
    results_array[*current_Index + offset[thread_Id] + 1] = output2;
    sign_array[*current_Index + offset[thread_Id]] = sign;
    sign_array[*current_Index + offset[thread_Id] + 1] = sign;
}