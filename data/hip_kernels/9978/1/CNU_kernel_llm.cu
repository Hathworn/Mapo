#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifndef _VNU_KERNEL_H_
#define _VNU_KERNEL_H_

#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#define database_character( index)  CUT_BANK_CHECKER(database_character, index)
#define temp_1( index)              CUT_BANK_CHECKER(temp_1,             index)
#define temp_2( index)              CUT_BANK_CHECKER(temp_2,             index)

#endif // #ifndef _VNU_KERNEL_H_

/*_____________________________________________Begin CN Kernel___________________________________________________*/
#ifndef _CNU_KERNEL_H_
#define _CNU_KERNEL_H_

#ifndef max
#define max( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#define database_character( index)  CUT_BANK_CHECKER(database_character, index)
#define temp_1( index)              CUT_BANK_CHECKER(temp_1,             index)
#define temp_2( index)              CUT_BANK_CHECKER(temp_2,             index)

#endif // #ifndef _CNU_KERNEL_H_

__global__ void CNU_kernel(short int* offset_array, short int* results_array)
{
    extern __shared__ short int shared_mem[];
    short int* offset_ptr = shared_mem;
    short int* thread_Id_ptr = offset_ptr + 1;
    short int* current_Index_ptr = thread_Id_ptr + 1;

    // Cache threadId and global memory index using shared memory
    *thread_Id_ptr = threadIdx.x;
    *current_Index_ptr = (blockIdx.x * blockDim.x + *thread_Id_ptr) * 3;

    // Load offset once per thread block
    *offset_ptr = offset_array[*current_Index_ptr];

    // CN Kernel Logic using registers for faster access
    short int input1 = results_array[*current_Index_ptr + *offset_ptr];
    short int input2 = results_array[*current_Index_ptr + *offset_ptr + 1];
    short int input3 = results_array[*current_Index_ptr + *offset_ptr + 2];

    short int min1 = 0;
    short int min2 = 0;
    short int agr = 1;

    if (input1 < 0) agr = -agr;
    if (input2 < 0) agr = -agr;
    if (input3 < 0) agr = -agr;

    // Simplified minimum evaluation logic
    if (abs(input1) <= abs(input2)) {
        min1 = input1;
        min2 = input2;
    } else {
        min1 = input2;
        min2 = input1;
    }

    if (abs(input3) <= abs(min1)) {
        min2 = min1;
        min1 = input3;
    } else if (abs(input3) <= abs(min2)) {
        min2 = input3;
    }

    // Write results back to global memory
    results_array[*current_Index_ptr + *offset_ptr] = min2 * agr;
    results_array[*current_Index_ptr + *offset_ptr + 1] = min1 * agr;
    results_array[*current_Index_ptr + *offset_ptr + 2] = min1 * agr;
}