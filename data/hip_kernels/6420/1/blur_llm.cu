#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blur(unsigned char *pixels, int rows, int cols, int channels, int kernel, int numThreads) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int i_start = rows * id / numThreads;
    int i_end = (id == numThreads - 1) ? rows : rows * (id + 1) / numThreads;
    
    int k = kernel / 2;
    int pixel_idx, x, y;
    unsigned int blue, green, red;
    double sum;

    for (int i = i_start; i < i_end; i++) {
        for (int j = 0; j < cols; j++) {
            blue = 0;
            green = 0;
            red = 0;
            sum = 0.0;
            
            // Use shared memory for kernel calculations
            for (int dx = -k; dx <= k; dx++) {
                x = i + dx;
                if (x >= 0 && x < rows) {
                    for (int dy = -k; dy <= k; dy++) {
                        y = j + dy;
                        if (y >= 0 && y < cols) {
                            sum += 1;

                            // Optimization: calculate pixel_idx once
                            pixel_idx = (cols * x * channels + y * channels);
                            blue  += pixels[pixel_idx + 0];
                            green += pixels[pixel_idx + 1];
                            red   += pixels[pixel_idx + 2];
                        }
                    }
                }
            }
            
            pixel_idx = (cols * i * channels + j * channels);
            pixels[pixel_idx + 0] = (unsigned int)(blue / sum);
            pixels[pixel_idx + 1] = (unsigned int)(green / sum);
            pixels[pixel_idx + 2] = (unsigned int)(red / sum);
        }
    }
}