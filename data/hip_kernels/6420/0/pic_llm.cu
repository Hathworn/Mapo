#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define ITER  10000000000  // Number of bins
#define NUMBLOCKS   13  // Number of thread blocks
#define NUMTHREADS 192  // Number of threads per block

int tid;
float pi;

// Kernel
__global__ void pic(float *sum, int nbin, float step, int nthreads, int nblocks) {
    int i;
    float x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Sequential thread index across the blocks
    float local_sum = 0.0f;  // Use register for local sum

    // Stride loop for work distribution
    for (i = idx; i < nbin; i += nthreads * nblocks) {
        x = (i + 0.5) * step;
        local_sum += 4.0f / (1.0f + x * x);
    }
    
    sum[idx] = local_sum;  // Store local sum in global memory
}