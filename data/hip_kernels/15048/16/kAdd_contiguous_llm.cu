#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Improved: Calculate the index and ensure it doesn't exceed numEls
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEls) {
        dest[idx] = a[idx] + b[idx];
    }
}