#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_dense_to_sparse_subtract(int n, const unsigned int *idx, float *src, float *trg) {
    // Optimize grid and block calculation for better performance
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use loop to process multiple elements per thread
    for (int i = id; i < n; i += stride) {
        atomicAdd(trg + idx[i], -src[i]);
    }
}