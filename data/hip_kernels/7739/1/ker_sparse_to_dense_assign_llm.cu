#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_sparse_to_dense_assign(int n, const unsigned int *idx, float *src, float *trg) {
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within array bounds
    if (id < n) {
        trg[id] = src[idx[id]];
    }
}