#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_dense_to_sparse_assign(int n, const unsigned int *idx, float *src, float *trg) {
    // Calculate the thread ID using faster intrinsic functions
    int id = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
    
    // Avoid out-of-bounds errors
    if (id < n) {
        trg[idx[id]] = src[id];
    }
}