#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_dense_to_sparse_block_add(int n, const unsigned *idx, int bsize, float* src, float *trg) {
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check that the threads don't exceed the bounds
    if (id < n * bsize) {
        int idx_base = idx[id / bsize] * bsize; // Calculate base index once
        int offset = id % bsize;                // Calculate offset once
        atomicAdd(trg + idx_base + offset, src[id]); // Perform atomic add
    }
}