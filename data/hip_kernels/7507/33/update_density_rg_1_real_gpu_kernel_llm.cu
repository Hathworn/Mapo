#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_density_rg_1_real_gpu_kernel(int size__, double const* __restrict__ psi_rg__, double wt__, double* __restrict__ density_rg__)
{
    int ir = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds
    if (ir < size__) {
        double p = psi_rg__[ir];

        // Use FMA operation for better performance
        density_rg__[ir] = fma(p, p * wt__, density_rg__[ir]);
    }
}