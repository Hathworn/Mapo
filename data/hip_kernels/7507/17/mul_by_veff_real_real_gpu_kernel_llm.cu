#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_by_veff_real_real_gpu_kernel(int nr__, double* buf__, double const* veff__)
{
    // Optimize block index calculation and check
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nr__) {
        buf__[idx] *= veff__[idx]; // Streamline multiplication operation
    }
}