#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved global kernel
__global__ void loop()
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    printf("This is iteration number %d\n", idx); // Use global index for iteration count
}