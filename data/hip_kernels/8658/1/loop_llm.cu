#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void loop()
{
    // Optimize printf by reducing the number of calls
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < gridDim.x * blockDim.x) {
        printf("This is iteration number %d\n", idx);
    }
}