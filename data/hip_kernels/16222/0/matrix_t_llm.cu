#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int CHUNK_SIZE = 32;
const int CHUNK_ROWS = 8;

__global__ void matrix_t(int* data, int* out, int* rows, int* cols) {
    __shared__ int chunk[CHUNK_SIZE][CHUNK_SIZE + 1]; // Avoid bank conflicts

    int x = blockIdx.x * CHUNK_SIZE + threadIdx.x;
    int y = blockIdx.y * CHUNK_SIZE + threadIdx.y;

    if (x < *cols && y < *rows) {
        // Efficiently load chunk of the matrix into shared memory
        for (int i = 0; i < CHUNK_SIZE; i += CHUNK_ROWS) {
            if (y + i < *rows) {
                chunk[threadIdx.y + i][threadIdx.x] = data[(y + i) * *cols + x];
            }
        }
    }
    __syncthreads();

    x = blockIdx.y * CHUNK_SIZE + threadIdx.x;
    y = blockIdx.x * CHUNK_SIZE + threadIdx.y;

    if (x < *rows && y < *cols) {
        // Write transposed chunk to the output matrix
        for (int i = 0; i < CHUNK_SIZE; i += CHUNK_ROWS) {
            if (y + i < *cols) {
                out[(y + i) * *rows + x] = chunk[threadIdx.x][threadIdx.y + i];
            }
        }
    }
}