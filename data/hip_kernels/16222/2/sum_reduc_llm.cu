#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_reduc(int* data, int* len, int* width) {
    int stride = blockDim.x * gridDim.x;
    int sum = 0;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < *len; i += stride) {
        sum += data[i];
    }
    __shared__ int shared_sum[256]; // Use shared memory for reduction
    int tx = threadIdx.x;
    shared_sum[tx] = sum;
    __syncthreads();

    // Reduce within block
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tx < s) {
            shared_sum[tx] += shared_sum[tx + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tx == 0) data[blockIdx.x] = shared_sum[0];
}