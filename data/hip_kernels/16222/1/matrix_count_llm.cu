#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_count(int* data, int* count, int* rows, int* cols) {
    __shared__ int chunk[CHUNK_SIZE][CHUNK_SIZE];

    int x = blockIdx.x * CHUNK_SIZE + threadIdx.x;
    int y = blockIdx.y * CHUNK_SIZE + threadIdx.y;
    int limit = CHUNK_SIZE / CHUNK_ROWS * CHUNK_ROWS;

    // Load data into shared memory efficiently
    for (int i = 0; i < limit; i += CHUNK_ROWS) {
        if ((y + i) < *rows && x < *cols) {
            chunk[threadIdx.y + i][threadIdx.x] = data[(y + i) * (*cols) + x];
        } else {
            chunk[threadIdx.y + i][threadIdx.x] = 0; // Handle out-of-bounds data
        }
    }
    __syncthreads();

    x = blockIdx.y * CHUNK_SIZE + threadIdx.x;
    y = blockIdx.x * CHUNK_SIZE + threadIdx.y;

    // Process data in shared memory
    for (int i = 0; i < CHUNK_SIZE; i += CHUNK_ROWS) {
        if (x < *cols && (y + i) < *rows) {
            if (chunk[threadIdx.x][threadIdx.y + i] == 1) {
                atomicAdd(count, 1); // Use atomic operation to update count
            }
        }
    }
}