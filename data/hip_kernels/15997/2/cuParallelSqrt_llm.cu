#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Constants used by the program
#define BLOCK_DIM 16
#define DEBUG 0

// Optimized kernel function
__global__ void cuParallelSqrt(float *dist, int width, int k) {
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Use single thread to process multiple elements if possible
    if (xIndex < width && yIndex < k) {
        int index = yIndex * width + xIndex;
        dist[index] = sqrt(dist[index]);
    }
}