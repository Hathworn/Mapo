#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Constants used by the program
#define BLOCK_DIM                      16
#define DEBUG                          0

__global__ void cuInsertionSort(float *dist, long *ind, int width, int height, int k){

    // Thread index in the x dimension
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform sorting only if within valid range
    if (xIndex < width) {

        // Initialize pointers and the max distance value
        float *p_dist = dist + xIndex;
        long  *p_ind = ind + xIndex;
        float max_dist = p_dist[0];
        p_ind[0] = 1;

        // Sort the first k elements
        for (int l = 1; l < k; l++) {
            long curr_row = l * width;
            float curr_dist = p_dist[curr_row];
            if (curr_dist < max_dist) {
                int i = l - 1;
                while (i >= 0 && p_dist[i * width] > curr_dist) {
                    i--;
                }
                i++; // Find insertion position
                for (int j = l; j > i; j--) {
                    // Shift elements to the right
                    p_dist[j * width] = p_dist[(j - 1) * width];
                    p_ind[j * width] = p_ind[(j - 1) * width];
                }
                // Insert current distance and update index
                p_dist[i * width] = curr_dist;
                p_ind[i * width] = l + 1;
            } else {
                p_ind[l * width] = l + 1;
            }
            max_dist = p_dist[curr_row]; // Update max distance
        }

        // Insert subsequent elements into the first k positions
        for (int l = k; l < height; l++) {
            float curr_dist = p_dist[l * width];
            if (curr_dist < max_dist) {
                int i = k - 1;
                while (i >= 0 && p_dist[i * width] > curr_dist) {
                    i--;
                }
                i++; // Find insertion position
                for (int j = k - 1; j > i; j--) {
                    // Shift elements to the right
                    p_dist[j * width] = p_dist[(j - 1) * width];
                    p_ind[j * width] = p_ind[(j - 1) * width];
                }
                // Insert current distance and update index
                p_dist[i * width] = curr_dist;
                p_ind[i * width] = l + 1;
                max_dist = p_dist[(k - 1) * width]; // Update max distance
            }
        }
    }
}