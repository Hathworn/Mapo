#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Constants used by the program
#define BLOCK_DIM                      16
#define DEBUG                          0

__global__ void cuComputeDistanceGlobal(float* A, int wA, float* B, int wB, int dim, float* AB) {
    // Shared memory for sub-matrix of A and B
    __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Calculate block indices
    int block_A = BLOCK_DIM * blockIdx.y; 
    int block_B = BLOCK_DIM * blockIdx.x; 

    // Other variables
    float ssd = 0;

    // Conditions for current block
    int valid_A = (block_A + ty < wA);
    int valid_B = (block_B + tx < wB);

    // Loop over all sub-matrices of A and B required to compute block sub-matrix
    for (int offset = 0; offset < dim; offset += BLOCK_DIM) {
        // Load shared memory
        if ((offset + ty) < dim) {
            shared_A[ty][tx] = (block_A + tx < wA) ? A[block_A + offset * wA + wA * ty + tx] : 0;
            shared_B[ty][tx] = (block_B + tx < wB) ? B[block_B + offset * wB + wB * ty + tx] : 0;
        }

        // Synchronize to ensure all threads have loaded
        __syncthreads();

        // Calculate ssd for current submatrix
        if (valid_A && valid_B) {
            for (int k = 0; k < BLOCK_DIM; ++k) {
                float tmp = shared_A[ty][k] - shared_B[tx][k];
                ssd += tmp * tmp;
            }
        }

        // Synchronize before the next load
        __syncthreads();
    }

    // Write result to global memory
    if (valid_A && valid_B) {
        AB[(block_A + ty) * wB + block_B + tx] = ssd;
    }
}