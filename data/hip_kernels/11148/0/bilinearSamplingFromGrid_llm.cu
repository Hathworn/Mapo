#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define real float

__device__ bool between(int value, int lowerBound, int upperBound)
{
    return (value >= lowerBound && value <= upperBound);
}

__device__ void getTopLeft(float x, int width, int& point, float& weight)
{
    float xcoord = (x + 1) * (width - 1) / 2;
    point = floor(xcoord);
    weight = 1 - (xcoord - point);
}

__global__ void bilinearSamplingFromGrid(float* inputImages_data, int inputImages_strideBatch, int inputImages_strideChannels,
                                         int inputImages_strideHeight, int inputImages_strideWidth, float* grids_data,
                                         int grids_strideBatch, int grids_strideYX, int grids_strideHeight, int grids_strideWidth,
                                         float* output_data, int output_strideBatch, int output_strideChannels, int output_strideHeight,
                                         int output_strideWidth, int inputImages_channels, int inputImages_height, int inputImages_width,
                                         int output_width)
{
    const int xOut = blockIdx.x * blockDim.y + threadIdx.y;
    const bool withinImageBounds = xOut < output_width;
    const bool withinGridBounds = blockIdx.x * blockDim.y + threadIdx.x / 2 < output_width;
    const int yOut = blockIdx.y;
    const int b = blockIdx.z;

    if (!withinImageBounds) return; // Bounds check for early exit

    const int width = inputImages_width;
    const int height = inputImages_height;

    __shared__ float gridData[32];
    if (threadIdx.y == 0 && withinGridBounds)
    {
        gridData[threadIdx.x] = grids_data[b * grids_strideBatch + yOut * grids_strideHeight + xOut * grids_strideWidth + threadIdx.x];
    }
    __syncthreads();

    float yf = gridData[threadIdx.y * 2];
    float xf = gridData[threadIdx.y * 2 + 1];

    int yInTopLeft, xInTopLeft;
    float yWeightTopLeft, xWeightTopLeft;
    getTopLeft(xf, inputImages_width, xInTopLeft, xWeightTopLeft);
    getTopLeft(yf, inputImages_height, yInTopLeft, yWeightTopLeft);

    const int outAddress = output_strideBatch * b + output_strideHeight * yOut + output_strideWidth * xOut;

    const int inTopLeftAddress = inputImages_strideBatch * b + inputImages_strideHeight * yInTopLeft + inputImages_strideWidth * xInTopLeft;
    const int inTopRightAddress = inTopLeftAddress + inputImages_strideWidth;
    const int inBottomLeftAddress = inTopLeftAddress + inputImages_strideHeight;
    const int inBottomRightAddress = inBottomLeftAddress + inputImages_strideWidth;

    float v = 0;
    float inTopLeft = 0, inTopRight = 0, inBottomLeft = 0, inBottomRight = 0;

    bool topLeftIsIn = between(xInTopLeft, 0, width - 1) && between(yInTopLeft, 0, height - 1);
    bool topRightIsIn = between(xInTopLeft + 1, 0, width - 1) && between(yInTopLeft, 0, height - 1);
    bool bottomLeftIsIn = between(xInTopLeft, 0, width - 1) && between(yInTopLeft + 1, 0, height - 1);
    bool bottomRightIsIn = between(xInTopLeft + 1, 0, width - 1) && between(yInTopLeft + 1, 0, height - 1);

    for (int t = threadIdx.x; t < inputImages_channels; t += blockDim.x)
    {
        if (topLeftIsIn) inTopLeft = inputImages_data[inTopLeftAddress + t];
        if (topRightIsIn) inTopRight = inputImages_data[inTopRightAddress + t];
        if (bottomLeftIsIn) inBottomLeft = inputImages_data[inBottomLeftAddress + t];
        if (bottomRightIsIn) inBottomRight = inputImages_data[inBottomRightAddress + t];

        v = xWeightTopLeft * yWeightTopLeft * inTopLeft
            + (1 - xWeightTopLeft) * yWeightTopLeft * inTopRight
            + xWeightTopLeft * (1 - yWeightTopLeft) * inBottomLeft
            + (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * inBottomRight;

        output_data[outAddress + t] = v;
    }
}