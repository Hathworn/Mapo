#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
extern "C"
__global__ void kernel(int* data, int size)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < size) // Ensure thread operates within data boundaries
    {
        data[id] = id;
    }
}