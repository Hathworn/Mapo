#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void advectParticles_OGL(float2 *part, float2 *v, int dx, int dy, float dt, int lb, size_t pitch) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;

    // Shared memory declaration for velocity vectors
    __shared__ float2 shared_v[blockDim.x];

    int p;
    float2 pterm, vterm;

    if (gtidx < dx) {
        for (p = 0; p < lb; p++) {
            int fi = gtidy + p;
            if (fi < dy) {
                int fj = fi * dx + gtidx;
                pterm = part[fj];

                int xvi = ((int)(pterm.x * dx));
                int yvi = ((int)(pterm.y * dy));

                // Use shared memory to reduce global memory access
                if (threadIdx.x == 0) {
                    shared_v[threadIdx.y] = *((float2*)((char*)v + yvi * pitch) + xvi);
                }
                __syncthreads();
                vterm = shared_v[threadIdx.y];
                __syncthreads();

                pterm.x += dt * vterm.x;
                pterm.x = pterm.x - (int)pterm.x;
                pterm.x += 1.f;
                pterm.x = pterm.x - (int)pterm.x;
                pterm.y += dt * vterm.y;
                pterm.y = pterm.y - (int)pterm.y;
                pterm.y += 1.f;
                pterm.y = pterm.y - (int)pterm.y;

                part[fj] = pterm;
            }
        }
    }
}