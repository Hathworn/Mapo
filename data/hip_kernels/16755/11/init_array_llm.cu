#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Using shared memory to improve access efficiency
    __shared__ int shared_factor;
    if (threadIdx.x == 0)
    {
        shared_factor = *factor;
    }
    __syncthreads();

    for (int i = 0; i < num_iterations; i++)
    {
        g_data[idx] += shared_factor; // Optimized: using shared memory
    }
}