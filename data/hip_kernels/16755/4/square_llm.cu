#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(int *array, int arrayCount)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Use warp-level operations to optimize performance
    if (idx < arrayCount) {
        int val = array[idx];
        val *= val;
        array[idx] = val;
    }
}