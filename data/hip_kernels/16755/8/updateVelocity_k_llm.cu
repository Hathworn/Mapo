#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateVelocity_k(float2 *v, float *vx, float *vy, int dx, int pdx, int dy, int lb, size_t pitch) {
    // Calculate thread index in global x and y dimensions
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;

    // Calculate scale factor once per kernel, as it is constant
    float scale = 1.f / (dx * dy);

    // Ensure thread is within domain bounds
    if (gtidx < dx) {
        for (int p = 0; p < lb; p++) {
            int fi = gtidy + p; // Calculate y domain index for this thread
            if (fi < dy) {
                int fjr = fi * pdx + gtidx; // Calculate flattened index in the arrays

                // Read vx and vy values
                float vxterm = vx[fjr];
                float vyterm = vy[fjr];

                // Normalize and prepare to store results in float2 type
                float2 nvterm;
                nvterm.x = vxterm * scale;
                nvterm.y = vyterm * scale;

                // Calculate pointer offset considering pitch and store result
                float2 *fj = (float2*)((char*)v + fi * pitch) + gtidx;
                *fj = nvterm;
            }
        }
    }
}