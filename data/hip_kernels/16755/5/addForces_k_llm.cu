#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addForces_k(float2 *v, int dx, int dy, int spx, int spy, float fx, float fy, int r, size_t pitch) {

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float2 *fj = (float2*)((char*)v + (ty + spy) * pitch) + tx + spx;

    // Preload fj for reuse and reduce global memory access
    float2 vterm = *fj;

    // Compute tx and ty minus r
    int txr = tx - r; 
    int tyr = ty - r;

    // Compute scaling factor s using precomputed txr and tyr
    float s = 1.f / (1.f + txr*txr*txr*txr + tyr*tyr*tyr*tyr);
    
    // Apply calculated force
    vterm.x += s * fx;
    vterm.y += s * fy;

    // Write updated force back to global memory
    *fj = vterm;
}