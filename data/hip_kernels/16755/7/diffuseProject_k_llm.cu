#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diffuseProject_k(float2 *vx, float2 *vy, int dx, int dy, float dt, float visc, int lb) {
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    
    // Optimize loop by merging data access and computation
    if (gtidx < dx) {
        for (int p = 0; p < lb; p++) {
            int fi = gtidy + p;
            if (fi < dy) {
                int fj = fi * dx + gtidx;
                
                float2 xterm = vx[fj];
                float2 yterm = vy[fj];

                // Calculate the index of the wavenumber
                int iix = gtidx;
                int iiy = (fi > dy / 2) ? (fi - dy) : fi;

                // Velocity diffusion calculation
                float kk = (float)(iix * iix + iiy * iiy);
                float diff = 1.f / (1.f + visc * dt * kk);
                xterm.x *= diff; 
                xterm.y *= diff;
                yterm.x *= diff; 
                yterm.y *= diff;

                // Velocity projection if k^2 is greater than zero
                if (kk > 0.f) {
                    float rkk = 1.f / kk;
                    float rkp = (iix * xterm.x + iiy * yterm.x);
                    float ikp = (iix * xterm.y + iiy * yterm.y);
                    
                    float iix_rkk = rkk * iix;
                    float iiy_rkk = rkk * iiy;

                    xterm.x -= iix_rkk * rkp;
                    xterm.y -= iix_rkk * ikp;
                    yterm.x -= iiy_rkk * rkp;
                    yterm.y -= iiy_rkk * ikp;
                }

                // Store back the updated terms
                vx[fj] = xterm;
                vy[fj] = yterm;
            }
        }
    }
}