#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAdd(const float* A, const float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure index i is within bounds and process data efficiently through coalesced memory access
    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}