#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using blockIdx and threadIdx
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index == 0) {
        *accuracy /= N; // Ensure correct execution by a single thread
    }
}