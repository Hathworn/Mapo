#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_max_kernel(float *d_out, const float *d_logLum, int size) {

    int tid  = threadIdx.x;                             // Local thread index
    int myId = blockIdx.x * blockDim.x + threadIdx.x;   // Global thread index

    extern __shared__ float temp[];

    // Load data to shared memory or set to a default minimum value if out of bounds
    temp[tid] = (myId < size) ? d_logLum[myId] : -FLT_MAX;  // Use defined constant for minimum float value
    __syncthreads(); // Ensure all loads are complete before reduction

    // Unroll the reduction loop for better performance
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            temp[tid] = fmaxf(temp[tid], temp[tid + s]);
        }
        __syncthreads(); // Ensure all memory operations are complete
    }

    // Write the result of the block's reduction to global output
    if (tid == 0) {
        d_out[blockIdx.x] = temp[0];
    }
}