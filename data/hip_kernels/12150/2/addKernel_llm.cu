#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index for all threads
    if (i < N) // Ensure within bounds for large data
    {
        c[i] = a[i] + b[i];
    }
}