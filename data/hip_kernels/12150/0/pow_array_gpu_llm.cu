#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_array_gpu(float *a, int power, int array_size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < array_size)
    {
        // Use intrinsics for power calculation to optimize
        a[idx] = powf(a[idx], power);
    }
}