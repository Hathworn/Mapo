#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int array_size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < array_size)
    {
        float value = a[idx]; // Load value once for reuse
        a[idx] = value * value; // Perform the computation
    }
}