#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel for parallel computation
__global__ void multiplyDet(double *matrix, double *determinant, int *n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int nn = *n;

    if (tid < nn) {
        // Use shared memory to accumulate partial products
        __shared__ double partialDet[1024];
        partialDet[tid] = matrix[tid * nn + tid];
        __syncthreads();

        // Use reduction to compute the determinant
        for (int stride = 1; stride < nn; stride *= 2) {
            if (tid % (2 * stride) == 0 && tid + stride < nn) {
                partialDet[tid] *= partialDet[tid + stride];
            }
            __syncthreads();
        }

        // Write the result back to the global memory
        if (tid == 0) {
            *determinant = partialDet[0];
        }
    }
}