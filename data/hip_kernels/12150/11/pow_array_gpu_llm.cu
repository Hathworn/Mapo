#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_array_gpu(float *a, int power, int array_size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < array_size)
    {
        float base = a[idx]; // Store base value
        float result = 1;
        for (int i = 0; i < power; ++i)
        {
            result *= base; // Use pre-stored value
        }
        a[idx] = result;
    }
}