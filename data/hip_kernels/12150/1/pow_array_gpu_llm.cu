#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pow_array_gpu(float *a, int power, int array_size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Only calculate if index is within array size range
    if (idx < array_size)
    {
        float base = a[idx];
        float result = 1.0f;

        // Use a more efficient method for power calculation
        for (int i = 0; i < power; ++i)
        {
            result *= base;
        }

        // Store the result back into the array
        a[idx] = result;
    }
}