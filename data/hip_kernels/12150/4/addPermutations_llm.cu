#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Macro to check HIP error codes
#define HIP_CHECK(cmd) do { \
  hipError_t result = cmd; \
  if (result != hipSuccess) { \
    printf("error: HIP error '%s' at %s:%d\n", hipGetErrorString(result), __FILE__, __LINE__); \
    exit(result); \
  } \
} while(0)

__global__ void addPermutations(double *determinant, const double *permutations, const int *n) {
    // Use shared memory and registers to improve performance
    extern __shared__ double shared_perm[];

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int nn = *n * *n - 1;
    double sum = 0.0;

    // Load permutations into shared memory
    if (idx < nn) {
        shared_perm[threadIdx.x] = permutations[idx];
    }
    __syncthreads();

    // Accumulate sum
    if (idx < nn) {
        sum = shared_perm[threadIdx.x];
    }

    // Atomic addition to prevent data race
    if (idx < nn) {
        atomicAdd(determinant, sum);
    }
}