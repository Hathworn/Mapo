#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_array_gpu(float *a, int power, int array_size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < array_size)
    {
        // Use a temporary variable to cache a[idx] to avoid redundant memory reads
        float base = a[idx];
        float result = 1.0f;

        // Use exponentiation by squaring for efficient computation
        int p = power;
        while (p > 0)
        {
            if (p % 2 == 1) // If power is odd
                result *= base;
            base *= base; // Square the base
            p /= 2; // Reduce power by half
        }
        a[idx] = result;
    }
}