#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addRows(double *matrix, int *d_i) {
    int i = *d_i;
    int n = blockDim.x + i;
    int id = n * (blockIdx.x + i + 1) + threadIdx.x + i;
    __shared__ double multiplier;

    // Move condition check outside to only set by thread 0
    if(threadIdx.x == 0) {
        multiplier = matrix[n * (blockIdx.x + 1 + i) + i] / matrix[n * i + i];
    }
    __syncthreads();

    // Use a separate variable to compute and access shared memory efficiently
    double matrix_val = matrix[n * i + threadIdx.x + i];

    // Reduce matrix access by using precomputed values
    matrix[id] -= matrix_val * multiplier;
}