#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use a grid-stride loop to improve parallel efficiency
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;

    for (int i = idx; i < gridSize; i += gridSize) {
        printf("Hello world! I'm thread %d\n", i);
    }
}