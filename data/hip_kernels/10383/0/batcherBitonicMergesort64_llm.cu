#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void batcherBitonicMergesort64(float *d_out, const float *d_in)
{
    // __shared__ memory for thread collaboration
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    
    // Load input data into shared memory
    sdata[tid] = d_in[tid];
    __syncthreads();

    // Perform bitonic sort
    for (int stage = 0; stage <= 5; stage++)
    {
        for (int substage = stage; substage >= 0; substage--)
        {
            int partner = tid ^ (1 << substage); // Calculate partner index in bitonic way
            // Ensure partner is within bounds
            if (partner < 64) {
                if (tid < partner) {
                    if (sdata[tid] > sdata[partner]) { 
                        // Swap elements to sort
                        float temp = sdata[tid];
                        sdata[tid] = sdata[partner];
                        sdata[partner] = temp;
                    }
                } else {
                    if (sdata[tid] < sdata[partner]) {
                        // Swap elements to sort
                        float temp = sdata[tid];
                        sdata[tid] = sdata[partner];
                        sdata[partner] = temp;
                    }
                }
            }
            __syncthreads(); // Synchronize before next step
        }
    }

    // Store sorted data back to global memory
    d_out[tid] = sdata[tid];
}