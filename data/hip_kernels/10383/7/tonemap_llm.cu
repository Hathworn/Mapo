#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tonemap( float* d_x, float* d_y, float* d_log_Y, float* d_cdf_norm, float* d_r_new, float* d_g_new, float* d_b_new, float  min_log_Y, float  max_log_Y, float  log_Y_range, int    num_bins, int    num_pixels_y, int    num_pixels_x )
{
    int2 image_index_2d = make_int2( blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y );
    int image_index_1d = num_pixels_x * image_index_2d.y + image_index_2d.x;

    // Only proceed if the pixel is within bounds
    if ( image_index_2d.x < num_pixels_x && image_index_2d.y < num_pixels_y )
    {
        float x = d_x[image_index_1d];
        float y = d_y[image_index_1d];
        float log_Y = d_log_Y[image_index_1d];
        
        // Compute bin index with clamped value for safety
        int bin_index = min( num_bins - 1, max(0, int((num_bins * (log_Y - min_log_Y)) / log_Y_range)));
        float Y_new = d_cdf_norm[bin_index];

        // Precompute reused values
        float inv_y = 1.0f / y;
        float Y_over_y = Y_new * inv_y;

        // Simplify color calculations by reusing calculated ratios
        float X_new = x * Y_over_y;
        float Z_new = (1 - x - y) * Y_over_y;

        // Efficient color transformations 
        float r_new = X_new * 3.2406f + Y_new * -1.5372f + Z_new * -0.4986f;
        float g_new = X_new * -0.9689f + Y_new * 1.8758f + Z_new * 0.0415f;
        float b_new = X_new * 0.0557f + Y_new * -0.2040f + Z_new * 1.0570f;

        d_r_new[image_index_1d] = r_new;
        d_g_new[image_index_1d] = g_new;
        d_b_new[image_index_1d] = b_new;
    }
}