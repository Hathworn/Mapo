#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
{
    const float normalization_constant = 1.f / d_input_cdf[n - 1];

    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread is within bounds
    if (global_index_1d < n)
    {
        // Directly compute and store normalized value
        d_output_cdf[global_index_1d] = d_input_cdf[global_index_1d] * normalization_constant;
    }
}