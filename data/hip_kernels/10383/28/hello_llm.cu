#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Improved: Use shared memory for reduced output contention in block
    __shared__ char message[50];
    sprintf(message, "Hello world! I'm a thread in block %d\n", blockIdx.x);
    printf("%s", message);
}