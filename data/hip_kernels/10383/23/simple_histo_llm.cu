#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;

    // Check bounds to prevent out-of-bounds memory access
    if (myId < BIN_COUNT) 
    {
        int myItem = d_in[myId];
        int myBin = myItem % BIN_COUNT;
        
        // Use atomic add to avoid race conditions
        atomicAdd(&(d_bins[myBin]), 1);
    }
}