#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void remove_redness_from_coordinates( const unsigned int*  d_coordinates, unsigned char* d_r, unsigned char* d_b, unsigned char* d_g, unsigned char* d_r_output, int num_coordinates, int num_pixels_y, int num_pixels_x, int template_half_height, int template_half_width )
{
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_index_1d < num_coordinates)
    {
        unsigned int image_index_1d = d_coordinates[global_index_1d]; // Use direct index
        ushort2 image_index_2d = make_ushort2(image_index_1d % num_pixels_x, image_index_1d / num_pixels_x);

        // Use shared memory for faster access
        __shared__ unsigned char shared_r[blockDim.x];
        __shared__ unsigned char shared_g[blockDim.x];
        __shared__ unsigned char shared_b[blockDim.x];

        for (int y = image_index_2d.y - template_half_height; y <= image_index_2d.y + template_half_height; y++)
        {
            for (int x = image_index_2d.x - template_half_width; x <= image_index_2d.x + template_half_width; x++)
            {
                int image_offset_index_1d_clamped = min(max(x, 0), num_pixels_x - 1) + min(max(y, 0), num_pixels_y - 1) * num_pixels_x;

                unsigned char g_value = d_g[image_offset_index_1d_clamped];
                unsigned char b_value = d_b[image_offset_index_1d_clamped];

                unsigned int gb_average = (g_value + b_value) / 2;

                d_r_output[image_offset_index_1d_clamped] = (unsigned char)gb_average;
            }
        }
    }
}