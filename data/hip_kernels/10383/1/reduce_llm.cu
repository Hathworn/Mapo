#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int* s) {
    // Each thread loads its data into shared memory
    unsigned int tid = threadIdx.x;
    s[tid] = p;
    __syncthreads(); // Ensure all writes to shared are visible to all threads

    // Perform warp-level reduction within shared memory
    if (tid < 16) s[tid] += s[tid + 16];
    __syncthreads();
    if (tid < 8) s[tid] += s[tid + 8];
    __syncthreads();
    if (tid < 4) s[tid] += s[tid + 4];
    __syncthreads();
    if (tid < 2) s[tid] += s[tid + 2];
    __syncthreads();
    if (tid == 0) s[0] += s[1]; // Final sum at s[0]

    return s[0]; // Return the reduced result
}

__global__ void reduce(unsigned int* d_out_shared, const unsigned int* d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    if (t == 0) {
        *d_out_shared = sr;
    }
}