#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth(float * v_new, const float * v) {
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x; // Correct thread index calculation
    int numThreads = gridDim.x * blockDim.x;
    if (myIdx < numThreads) { // Boundary check within the valid range
        int myLeftIdx = (myIdx == 0) ? 0 : myIdx - 1; // Safe access to left index
        int myRightIdx = (myIdx == (numThreads - 1)) ? numThreads - 1 : myIdx + 1; // Safe access to right index
        float myElt = v[myIdx];
        float myLeftElt = v[myLeftIdx];
        float myRightElt = v[myRightIdx];
        v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt; // Update new value
    }
}