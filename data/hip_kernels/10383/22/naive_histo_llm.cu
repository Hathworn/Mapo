#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naive_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    extern __shared__ int shared_bins[]; // Use shared memory for bin counting
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;

    if (threadIdx.x < BIN_COUNT) // Initialize shared memory bins
        shared_bins[threadIdx.x] = 0;
    __syncthreads();

    atomicAdd(&shared_bins[myBin], 1); // Use atomicAdd on shared memory
    __syncthreads();

    if (threadIdx.x < BIN_COUNT) // Increment the global bins
        atomicAdd(&d_bins[threadIdx.x], shared_bins[threadIdx.x]);
}