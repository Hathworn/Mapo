#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_ClassNLLCriterion_updateGradInput_kernel(float *gradInput, float *target, int nframe, int ndim, float grad, int ntarget, float* weights, bool apply_weights) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop to handle more elements per thread
    for (int i = idx; i < nframe; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < ntarget; ++j) {
            int t = (int)target[i * ntarget + j] - 1;
            if (t >= 0) { // Check valid index
                if (apply_weights) {
                    gradInput[i * ndim + t] = grad * weights[t]; // Weighted gradient
                } else {
                    gradInput[i * ndim + t] = grad; // Regular gradient
                }
            }
        }
    }
}