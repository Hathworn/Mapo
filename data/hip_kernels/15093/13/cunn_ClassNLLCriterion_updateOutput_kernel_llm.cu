#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NTHREADS 256  // Define the number of threads for shared memory

__global__ void cunn_ClassNLLCriterion_updateOutput_kernel(float *output, float *input, float *target, int nframe, int ndim, int sizeAverage, int ntarget, float* weights, bool apply_weights) {
    __shared__ float shInputs[NTHREADS];
    int i, j, t;
    
    // Initialize shared memory
    shInputs[threadIdx.x] = 0.0f;

    // Calculate contribution for each thread
    for (i = threadIdx.x; i < nframe; i += blockDim.x) {
        for (j = 0; j < ntarget; ++j) {
            t = static_cast<int>(target[i * ntarget + j]) - 1;
            if (t >= 0) {
                if (apply_weights) {
                    shInputs[threadIdx.x] += (input[i * ndim + t] * weights[t]);
                } else {
                    shInputs[threadIdx.x] += input[i * ndim + t];
                }
            }
        }
    }
    
    __syncthreads();

    // Reduce results from all threads
    if (threadIdx.x == 0) {
        float sum = 0.0f;
        for (i = 0; i < blockDim.x; ++i) {
            sum += shInputs[i];
        }
        if (sizeAverage)
            *output = -(sum / nframe);
        else
            *output = -sum;
    }
}