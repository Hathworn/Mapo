#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void atomicadaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w)
{
    // Compute offsets based on thread/block ID
    int o = blockIdx.x;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += o * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // Compute gradInput
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        int y_start = static_cast<int>(yy * input_h / output_h);  // Precompute division

        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            int x_start = static_cast<int>(xx * input_w / output_w);  // Precompute division

            float *ptr_gradInput = gradInput + y_start * input_w + x_start;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput;

            int argmax_x = static_cast<int>(indices_x[yy * output_w + xx]) - 1;  // Reduce dereferencing
            int argmax_y = static_cast<int>(indices_y[yy * output_w + xx]) - 1;  // Reduce dereferencing

            // Atomic add since different threads could update the same variable
            atomicAdd(&(ptr_gradInput[argmax_x + argmax_y * input_w]), z);
        }
    }
}