#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y * gridDim.y;

    // select input/output plane
    gradOutput = gradOutput + o * output_w * output_h;
    gradInput = gradInput + i * input_w * input_h;

    // get weight
    float the_weight = weight[k];

    // cache blockDim.x calculation
    int input_w_stride = dH * input_w;

    // compute gradInput
    for (int yy = yy_start; yy < yy_end; yy += yy_step) {
        for (int xx = xx_start; xx < xx_end; xx += xx_step) {
            float *ptr_gradInput = gradInput + yy * input_w_stride + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = (*ptr_gradOutput) * the_weight;

            // optimize inner loop by avoiding recalculating pointers
            float *ptr_input_row = ptr_gradInput;
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    atomicAdd(&(ptr_input_row[kx]), z);
                }
                ptr_input_row += input_w;
            }
        }
    }
}