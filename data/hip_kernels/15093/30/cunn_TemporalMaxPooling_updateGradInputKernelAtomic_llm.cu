#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate offsets once outside the loop to reduce repeated calculations in the loop
    int timeIndex = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    if (timeIndex < output_w) {
        float *gradInput_data = gradInput + blockIdx.x * input_w * input_n + timeIndex * input_n * dW;
        float *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + timeIndex * input_n;
        float *indices_data = indices + blockIdx.x * output_w * input_n + timeIndex * input_n;

        // Use register to hold the offset index for atomicAdd to minimize memory access
        int offset;
        for (int feat = 0; feat < input_n; ++feat) {
            offset = (int)indices_data[feat] * input_n + feat;
            atomicAdd(&gradInput_data[offset], gradOutput_data[feat]);
        }
    }
}