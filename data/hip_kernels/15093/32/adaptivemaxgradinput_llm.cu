#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w)
{
    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;
    
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // Compute gradInput
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        int y_start = (int)((yy * input_h) / output_h);

        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            int x_start = (int)((xx * input_w) / output_w);

            float *ptr_gradInput = gradInput + y_start * input_w + x_start;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float *ptr_ind_x = indices_x + yy * output_w + xx;
            float *ptr_ind_y = indices_y + yy * output_w + xx;
            float z = *ptr_gradOutput;

            int argmax_x = static_cast<int>(*ptr_ind_x) - 1;
            int argmax_y = static_cast<int>(*ptr_ind_y) - 1;

            ptr_gradInput[argmax_x + argmax_y * input_w] += z;
        }
    }
}