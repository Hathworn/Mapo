#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x = ii / (d2 * d3);
    int y = (ii / d3) % d2;
    int z = ii % d3;
    y = y * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    return ((x * d1 + y) * d2 + z);
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x = ii / (d2 * d3);
    int y = (ii / d3) % d2;
    int z = ii % d3;
    y /= scale_factor;
    z /= scale_factor;
    d2 /= scale_factor;
    return ((x * d1 + y) * d2 + z);
}

__global__ void downscale(float* gradInput_data, float* gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    // Calculate output offset
    long ii = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.y * gridDim.x * blockDim.y + threadIdx.y * gridDim.x * blockDim.x;
    if (ii >= no_elements) return; // Early return for bounds check
    
    // Loop unrolling optimization
    #pragma unroll
    for (int i = 0; i < scale_factor; i++) {
        #pragma unroll
        for (int j = 0; j < scale_factor; j++) {
            int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
            gradInput_data[ii] += gradOutput_data[ipidx];
        }
    }
}