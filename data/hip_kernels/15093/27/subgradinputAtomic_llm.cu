#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW) {
    // Get thread/block identifiers and compute output dimensions
    int o = blockIdx.x;
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;
    int yy_start = blockIdx.y * blockDim.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Adjust gradOutput and gradInput pointers for current output/input plane respectively
    gradOutput += o * output_w * output_h;
    gradInput += o * input_w * input_h;

    // Iterate over output height
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        // Iterate over output width
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            // Compute the pointer locations for gradInput and gradOutput
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput;

            // Accumulate gradient over kernel height and width
            // Reduce repeated computations and improve memory access pattern
            float increment = z / float(kW * kH);
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    atomicAdd(&(ptr_gradInput[kx]), increment); // Atomic operation keeps consistency
                }
                ptr_gradInput += input_w; // Move to the next row down on gradInput
            }
        }
    }
}