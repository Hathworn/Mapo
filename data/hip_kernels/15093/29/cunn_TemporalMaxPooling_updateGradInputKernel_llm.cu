#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate the initial index positions using block and thread indices
    int time_idx = threadIdx.x + blockIdx.y * blockDim.x;
    if (time_idx >= output_w) return;  // Early exit if out of bounds

    int batch_offset = blockIdx.x * input_w * input_n;
    float *gradInput_data = gradInput + batch_offset + time_idx * input_n * dW;
    float *gradOutput_data = gradOutput + batch_offset + time_idx * input_n;
    float *indices_data = indices + batch_offset + time_idx * input_n;

    // Loop over all features
    for (int feat = 0; feat < input_n; ++feat) {
        int index = static_cast<int>(indices_data[feat]) * input_n + feat;  // Compute index once
        atomicAdd(&gradInput_data[index], gradOutput_data[feat]);  // Use atomicAdd to prevent race conditions
    }
}
```
