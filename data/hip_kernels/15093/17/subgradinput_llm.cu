#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradinput(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;
    
    // compute start and step indices
    int xx = threadIdx.x;
    int yy = blockDim.y * blockIdx.y + threadIdx.y;
    int xx_step = blockDim.x;
    int yy_step = blockDim.y * gridDim.y;

    // select input/output plane
    gradOutput = gradOutput + o * output_w * output_h;
    gradInput = gradInput + i * input_w * input_h;

    // get weight
    float the_weight = weight[k];

    // compute gradInput
    for(; yy < output_h; yy += yy_step) {
        for(int x = xx; x < output_w; x += xx_step) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + x * dW;
            float z = gradOutput[yy * output_w + x] * the_weight;
            for(int ky = 0; ky < kH; ky++) {
                #pragma unroll    // Enable loop unrolling for kW loop
                for(int kx = 0; kx < kW; kx++) {
                    ptr_gradInput[kx] += z;
                }
                ptr_gradInput += input_w;
            }
        }
    }
}