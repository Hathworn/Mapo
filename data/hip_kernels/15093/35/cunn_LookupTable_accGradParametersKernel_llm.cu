#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_LookupTable_accGradParametersKernel(float *input, float *indices, float *gradOutput, float *gradWeight, float *count, float defaultScale, long numel, long stride) {

    // Efficiently calculate idx for warp-level parallelism
    int idx = blockIdx.x * blockDim.y + threadIdx.y;

    // Handle boundary conditions and skip redundant computations
    if (idx >= numel || (idx != 0 && input[idx] == input[idx - 1])) return;

    const int SZ = 4; // Define grain size for each thread

    do {
        // Calculate starting feature for the current thread
        const int startFeature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
        const int weightRow = ((int) input[idx] - 1) * stride;
        const int gradOutputRow = ((int) indices[idx] - 1) * stride;
        const float scale = count ? defaultScale / count[idx] : defaultScale;

        float gradient[SZ];
        float weight[SZ];

        // Preload gradient and weight values using unrolled loop
        #pragma unroll
        for (int ii = 0; ii < SZ; ii++) {
            int featureDim = startFeature + ii * WARP_SIZE;
            if (featureDim < stride) {
                gradient[ii] = gradOutput[gradOutputRow + featureDim];
                weight[ii] = gradWeight[weightRow + featureDim];
            }
        }

        // Perform scaled addition operation using unrolled loop
        #pragma unroll
        for (int ii = 0; ii < SZ; ii++) {
            weight[ii] += gradient[ii] * scale;
        }

        // Write back computed weights using unrolled loop
        #pragma unroll
        for (int ii = 0; ii < SZ; ii++) {
            int featureDim = startFeature + ii * WARP_SIZE;
            if (featureDim < stride) {
                gradWeight[weightRow + featureDim] = weight[ii];
            }
        }

        idx++;
    } while (idx < numel && input[idx] == input[idx - 1]);
}