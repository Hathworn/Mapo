#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y * gridDim.y;

    // select input/output plane
    output = output + o * output_w * output_h;
    input = input + i * input_w * input_h;

    // For all output pixels...
    for (int yy = yy_start; yy < yy_end; yy += yy_step) {
        for (int xx = xx_start; xx < xx_end; xx += xx_step) {
            // Compute the mean of the input image...
            float sum = 0;
            int src_off_base = yy * dH * input_w + xx * dW;
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    // Calculate input index using offsets
                    int src_off = src_off_base + ky * input_w + kx;
                    sum += input[src_off];
                }
            }
            // Calculate output index and update output
            int dst_off = yy * output_w + xx;
            output[dst_off] = sum / float(kW * kH);  // Calculate mean
        }
    }
}