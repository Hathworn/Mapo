#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_MSECriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, float norm, int nframe, int dim) {
    int k = blockIdx.x;
    float *gradInput_k = gradInput + k * dim;
    float *input_k = input + k * dim;
    float *target_k = target + k * dim;

    int i = threadIdx.x;

    // Using stride loop to improve memory coalescing and utilize more threads
    for (; i < dim; i += blockDim.x) {
        gradInput_k[i] = norm * (input_k[i] - target_k[i]);
    }
}