#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_SoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int nframe, int dim)
{
    __shared__ float buffer[SOFTMAX_THREADS];
    int k = blockIdx.x;
    float *gradInput_k = gradInput + k * dim;
    float *output_k = output + k * dim;
    float *gradOutput_k = gradOutput + k * dim;

    int i_start = threadIdx.x;
    int i_step = blockDim.x;

    // Efficient sum calculation using warp-level synchronization.
    float sum_partial = 0.0f;
    for (int i = i_start; i < dim; i += i_step) {
        sum_partial += gradOutput_k[i] * output_k[i];
    }
    buffer[threadIdx.x] = sum_partial;
    __syncthreads();

    // Warp reduction
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        if (threadIdx.x < offset) {
            buffer[threadIdx.x] += buffer[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Update gradInput using the computed sum
    float sum_k = buffer[0];
    for (int i = i_start; i < dim; i += i_step) {
        gradInput_k[i] = output_k[i] * (gradOutput_k[i] - sum_k);
    }
}