#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradinput(float *gradInput, float *gradOutput, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // compute output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // compute offsets based on thread/block ID
    int o = blockIdx.x;  // output plane
    int i = o;  // input plane

    int xx = blockDim.x * blockIdx.y + threadIdx.x; // flatten 2D grid to 1D for xx
    int yy = blockDim.y * blockIdx.z + threadIdx.y; // flatten 2D grid to 1D for yy

    if (xx < output_w && yy < output_h) {
        // calculate offsets to gradInput and gradOutput arrays
        float* ptr_gradInput = gradInput + i * input_w * input_h + yy * dH * input_w + xx * dW;
        float* ptr_gradOutput = gradOutput + o * output_w * output_h + yy * output_w + xx;
        float z = *ptr_gradOutput;
        
        // compute gradInput within kernel limits
        for (int ky = 0; ky < kH; ++ky) {
            for (int kx = 0; kx < kW; ++kx) {
                ptr_gradInput[kx] += z / (float)(kW * kH);
            }
            ptr_gradInput += input_w;
        }
    }
}