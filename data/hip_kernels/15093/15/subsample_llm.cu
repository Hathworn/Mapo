#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Determine output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Calculate the output pixel coordinates this thread will handle
    int xx = threadIdx.x + blockIdx.x * blockDim.x;
    int yy = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure the thread is operating within bounds
    if (xx < output_w && yy < output_h) {
        int o = blockIdx.x;
        int i = o;
        int k = blockIdx.x % input_n;

        // Position output pointer
        float* output_ptr = output + o * output_w * output_h + yy * output_w + xx;
        // Position input pointer
        float* input_ptr = input + i * input_w * input_h + yy * dH * input_w + xx * dW;

        // Load weight and bias
        float the_weight = weight[k];
        float the_bias = bias[k];

        // Calculate the sum for the kernel
        float sum = 0.0f;
        for (int ky = 0; ky < kH; ++ky) {
            for (int kx = 0; kx < kW; ++kx) {
                sum += input_ptr[ky * input_w + kx];
            }
        }

        // Update the output with the weighted sum and bias
        *output_ptr = the_weight * sum + the_bias;
    }
}
```
