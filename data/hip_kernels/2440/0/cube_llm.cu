#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void cube(float* d_out, const float* __restrict__ d_in) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Calculate global index
    float f = d_in[idx];
    d_out[idx] = f * f * f;
}