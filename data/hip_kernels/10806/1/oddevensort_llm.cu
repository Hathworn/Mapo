#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void oddevensort(int *in, int *out, int size)
{
    __shared__ bool swappedodd;
    __shared__ bool swappedeven;
    int temp;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    bool oddeven = true;

    while (true)
    {
        __syncthreads(); // Synchronize at the start of the loop
        if (oddeven)
        {
            swappedodd = false;
            __syncthreads();

            if (idx < size / 2)
            {
                int i = 2 * idx;
                if (i + 1 < size && in[i] > in[i + 1])
                {
                    temp = in[i];
                    in[i] = in[i + 1];
                    in[i + 1] = temp;
                    swappedodd = true;
                }
            }
            __syncthreads();
        }
        else
        {
            swappedeven = false;
            __syncthreads();

            if (idx < (size / 2) - 1)
            {
                int i = 2 * idx + 1;
                if (i + 1 < size && in[i] > in[i + 1])
                {
                    temp = in[i];
                    in[i] = in[i + 1];
                    in[i + 1] = temp;
                    swappedeven = true;
                }
            }
            __syncthreads();
        }

        if (!swappedodd && !swappedeven)
        {
            break;
        }
        oddeven = !oddeven;
    }

    // Write the sorted output
    if (idx < size)
    {
        out[idx] = in[idx];
    }
}