#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute(int *v1, int *v2, int *v3, int N) {
    int idx = N * blockIdx.x + threadIdx.x; // Calculate the global index once

    if(blockIdx.x == 2) {
        // Simplify the indexing calculations
        int idx_m2 = idx - 2 * N;
        int idx_m1 = idx - N;
        v3[idx] = v1[idx_m2] * v2[idx_m1] - v1[idx_m1] * v2[idx_m2];
    } else if(blockIdx.x == 1) {
        // Similar simplification for blockIdx.x == 1
        int idx_p1 = idx + N;
        v3[idx] = v1[idx_p1] * v2[idx_m1] - v1[idx_m1] * v2[idx_p1];
    } else {
        // Similar simplification for blockIdx.x == 0
        int idx_p1 = idx + N;
        int idx_p2 = idx + 2 * N;
        v3[idx] = v1[idx_p1] * v2[idx_p2] - v2[idx_p1] * v1[idx_p2];
    }
}