#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMul(float *A, float *B, float *C, int n)
{
    // Compute row and column indices
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    // Initialize sum
    float sum = 0.0f;

    // Perform the computation only if within matrix bounds
    if (row < n && col < n) {
        // Use shared memory for a and b submatrices
        __shared__ float A_shared[32][32];
        __shared__ float B_shared[32][32];

        for (int i = 0; i < n; i += 32) {
            // Collaboratively load A and B tiles into shared memory
            A_shared[threadIdx.y][threadIdx.x] = A[row*n + i + threadIdx.x];
            B_shared[threadIdx.y][threadIdx.x] = B[(i+threadIdx.y)*n + col];
            __syncthreads(); // Ensure all threads have loaded the tiles

            // Calculate partial sum for the tile
            for (int j = 0; j < 32; ++j) {
                sum += A_shared[threadIdx.y][j] * B_shared[j][threadIdx.x];
            }
            __syncthreads(); // Ensure all threads have completed the compute
        }

        // Write the result to the output matrix
        C[row*n + col] = sum;

        // Debug print
        printf("\n Block[%d][%d] : Thread[%d][%d] : Product = %.2f\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, sum);
    }
}