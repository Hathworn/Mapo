#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Improved 1D grid-stride loop to handle larger arrays efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (; i < n; i += stride) {
        float weighted_a = s[i] * a[i];
        float weighted_b = (1 - s[i]) * (b ? b[i] : 0);
        c[i] = weighted_a + weighted_b; // Removed redundant calculations
    }
}