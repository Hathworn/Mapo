#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate the index within the grid
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N) { // Ensure within bounds
        // Calculate indices using modulus and division for batching 
        int in_s = index % spatial;
        int in_c = (index / spatial) % layers;
        int b = (index / spatial) / layers;

        // Calculate flat index positions for x and out arrays
        int i1 = b * layers * spatial + in_c * spatial + in_s;
        int i2 = b * layers * spatial + in_s * layers + in_c;

        // Use ternary operator to simplify assignment
        out[forward ? i2 : i1] = x[forward ? i1 : i2];
    }
}