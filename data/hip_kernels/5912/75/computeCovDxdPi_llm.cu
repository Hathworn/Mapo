#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeCovDxdPi(int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *inverse_covariance, int voxel_num, double gauss_d1, double gauss_d2, double *point_gradients, double *cov_dxd_pi, int valid_voxel_num)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int row = blockIdx.y;
    int col = blockIdx.z;

    if (row < 3 && col < 6) {
        double *icov_start = inverse_covariance + row * 3 * voxel_num;
        double *cov_dxd_pi_start = cov_dxd_pi + (row * 6 + col) * valid_voxel_num;
        double *pg_start = point_gradients + col * valid_points_num;

        // Move to register for faster access
        double icov0[voxel_num], icov1[voxel_num], icov2[voxel_num];
        for (int k = 0; k < voxel_num; ++k) {
            icov0[k] = icov_start[k];
            icov1[k] = icov_start[k + voxel_num];
            icov2[k] = icov_start[k + 2 * voxel_num];
        }

        for (int i = id; i < valid_points_num; i += stride) {
            double pg0 = pg_start[i];
            double pg1 = pg_start[i + 6 * valid_points_num];
            double pg2 = pg_start[i + 12 * valid_points_num];

            for (int j = starting_voxel_id[i]; j < starting_voxel_id[i + 1]; j++) {
                int vid = voxel_id[j];
                cov_dxd_pi_start[j] = icov0[vid] * pg0 + icov1[vid] * pg1 + icov2[vid] * pg2;
            }
        }
    }
}