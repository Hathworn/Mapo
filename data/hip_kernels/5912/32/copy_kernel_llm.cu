#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Grid stride loop to efficiently cover all elements
    for (; i < N; i += blockDim.x * gridDim.x)
    {
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
    }
}