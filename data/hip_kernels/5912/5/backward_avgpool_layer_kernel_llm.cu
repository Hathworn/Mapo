#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Use flat indexing for block
    if(id >= n * c) return;  // Adjust range check for combined c and n

    int k = id % c;
    int b = id / c;  // Directly calculate batch index

    float out_val = out_delta[k + c*b] / (w*h);  // Precompute common value

    for(int i = 0; i < w*h; ++i) {
        int in_index = i + h*w*(k + b*c);
        atomicAdd(&in_delta[in_index], out_val);  // Use atomicAdd for safer addition
    }
}