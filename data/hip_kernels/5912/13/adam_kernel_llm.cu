#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    // Precompute values outside loop
    float B1_t = 1.f - powf(B1, t);
    float B2_t = 1.f - powf(B2, t);

    // Use local variables for repeated calculations
    float m_val = m[index];
    float v_val = v[index];

    float mhat = m_val / B1_t;
    float vhat = v_val / B2_t;

    x[index] = x[index] + rate * mhat / (sqrtf(vhat) + eps);
}