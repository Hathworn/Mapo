#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    // Calculate dimensions outside the loop for efficiency
    int h = (in_h + 2*pad) / stride;
    int w = (in_w + 2*pad) / stride;
    int c = in_c;

    // Compute the global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Return if the thread index is out of bounds
    if(id >= n) return;

    // Calculate indices for accessing data
    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    // Calculate offsets
    int w_offset = -pad;
    int h_offset = -pad;

    // Calculate the output index
    int out_index = j + w * (i + h * (k + c * b));
    float max = -INFINITY;
    int max_i = -1;

    // Use narrower scope for loop variables
    for(int l = 0; l < size; ++l){
        for(int m = 0; m < size; ++m){
            int cur_h = h_offset + i * stride + l;
            int cur_w = w_offset + j * stride + m;
            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            int valid = (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w);

            // Compare and update maximum value
            if (valid) {
                float val = input[index];
                if (val > max) {
                    max = val;
                    max_i = index;
                }
            }
        }
    }

    // Store the results
    output[out_index] = max;
    indexes[out_index] = max_i;
}