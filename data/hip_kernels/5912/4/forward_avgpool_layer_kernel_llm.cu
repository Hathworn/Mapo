#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    // Using flat thread index for better coalescence
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Calculate output index
    int k = id % c;
    int b = id / c;
    int out_index = k + c * b;

    // Initialize output accumulator
    float sum = 0.0f;

    // Use threadIdx.x to offset contributions, loop unroll for performance
    for (int i = threadIdx.x; i < w * h; i += blockDim.x) {
        int in_index = i + h * w * (k + b * c);
        sum += input[in_index];
    }

    // Use shared memory for reduction if blockDim.x > 1
    __shared__ float shared_sum[1024]; // Support up to 1024 threads per block
    shared_sum[threadIdx.x] = sum;
    __syncthreads();

    // Reduction step
    if (threadIdx.x == 0) {
        float total_sum = 0.0f;
        for (int j = 0; j < blockDim.x; ++j) {
            total_sum += shared_sum[j];
        }
        output[out_index] = total_sum / (w * h);
    }
}