#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    // Calculate linear thread index within the grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Precompute the variance term outside the loop for efficiency
    float variance_term = -1.f / sqrtf(variance[i] + .00001f);

    // Initialize mean_delta[i] directly in the kernel
    float sum = 0.0f;

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            // Accumulate delta values in local variable
            sum += delta[index];
        }
    }
    
    // Efficiently update mean_delta by utilizing precomputed variance term
    mean_delta[i] = sum * variance_term;
}