#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate the global thread index more clearly
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a stride to allow for larger concurrency
    int stride = blockDim.x * gridDim.x;

    // Loop over entire array in strides
    for (; i < N; i += stride) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}