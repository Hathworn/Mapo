#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumScore(double *score, int full_size, int half_size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory to minimize global memory access
    extern __shared__ double sharedScore[];
    
    for (int tid = threadIdx.x; tid < half_size; tid += blockDim.x) {
        sharedScore[tid] = score[tid];
        __syncthreads(); // Ensure all threads have updated shared memory

        if (tid + half_size < full_size) {
            sharedScore[tid] += score[tid + half_size];
        }
        __syncthreads(); // Ensure all updates are completed before writing back

        score[tid] = sharedScore[tid];
    }
}