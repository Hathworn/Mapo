#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Efficient check to stay within bounds
    if (i < n) {
        binary[i] = (x[i] >= 0) ? 1 : -1;
    }
}