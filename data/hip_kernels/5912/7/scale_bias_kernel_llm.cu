#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    // Calculate global thread index
    int batch = blockIdx.z; 
    int filter = blockIdx.y; 
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    // Process elements only within the valid range
    if (offset < size) {
        int index = (batch * n + filter) * size + offset;
        output[index] *= biases[filter]; // Apply bias scaling
    }
}