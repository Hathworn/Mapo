#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void col2im_gpu_kernel(const int n, const float* data_col, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_im) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    // Use shared memory for better performance (assumes small ksize)
    __shared__ float col_cache[256]; // Adjust size as necessary
    for(; index < n; index += blockDim.x*gridDim.x){
        float val = 0;
        int w = index % width + pad;
        int h = (index / width) % height + pad;
        int c = index / (width * height);
        
        // Compute the start and end of the output
        int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
        int w_col_end = min(w / stride + 1, width_col);
        int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
        int h_col_end = min(h / stride + 1, height_col);
        
        // Equivalent implementation using computed offsets
        int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col;
        int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
        int coeff_w_col = (1 - stride * height_col * width_col);
        
        for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
            // Load into shared memory
            for (int w_col = threadIdx.x; w_col < width_col; w_col += blockDim.x) {
                col_cache[w_col] = data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
            }
            __syncthreads();
            
            for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
                val += col_cache[w_col];
            }
            __syncthreads();
        }
        data_im[index] += val;
    }
}