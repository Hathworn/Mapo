#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (i < n) {
        float t = truth[i];
        float p = pred[i];

        // Compute error
        error[i] = -t * logf(p + 1e-7f) - (1 - t) * logf(1 - p + 1e-7f);

        // Compute delta
        delta[i] = t - p;
    }
}
```
