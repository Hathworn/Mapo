#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    int w_offset = -(size / 2);
    int h_offset = -(size / 2);
    int out_index = j + w * (i + h * (k + c * b));

    // Pre-calculate indices to reduce redundant computations inside the loops
    float out_value = x[out_index];

    for(int l = 0; l < size; ++l){
        int cur_h = h_offset + i + l;
        if(cur_h >= 0 && cur_h < h) {  // Only proceed if cur_h is valid
            for(int m = 0; m < size; ++m){
                int cur_w = w_offset + j + m;
                if(cur_w >= 0 && cur_w < w) {  // Only proceed if cur_w is valid
                    int index = cur_w + w * (cur_h + h * (k + b * c));
                    delta[out_index] += rate * (x[index] - out_value);
                }
            }
        }
    }
}