#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    // Use shared memory to reduce global memory accesses
    extern __shared__ float s_data[];
    s_data[threadIdx.x] = 0.0f;

    // Accumulate values in shared memory
    for(int k = threadIdx.x; k < n; k += blockDim.x) {
        s_data[threadIdx.x] += x[k * groups + i];
    }

    // Reduce within block
    __syncthreads();
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write result from shared memory to global memory
    if (threadIdx.x == 0) {
        sum[i] = s_data[0];
    }
}