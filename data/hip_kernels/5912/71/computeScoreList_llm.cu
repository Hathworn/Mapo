#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeScoreList(int *starting_voxel_id, int *voxel_id, int valid_points_num, double *e_x_cov_x, double gauss_d1, double *score)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Ensure thread does not exceed number of valid points
    if (id < valid_points_num) {
        double score_inc = 0;

        // Optimize loop to reduce shared memory divergence
        int start_vid = starting_voxel_id[id];
        int end_vid = starting_voxel_id[id + 1];

        for (int vid = start_vid; vid < end_vid; vid++) {
            double tmp_ex = e_x_cov_x[vid];

            // Use branch-free method for nan check
            if (!(tmp_ex > 1 || tmp_ex < 0 || tmp_ex != tmp_ex)) {
                score_inc -= gauss_d1 * tmp_ex;
            }
        }

        score[id] = score_inc;
    }
}