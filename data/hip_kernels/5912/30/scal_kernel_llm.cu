#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Use one dimensional grid index for simpler computation
    int stride = gridDim.x * blockDim.x; // Calculate stride for loop unrolling
    for(int i = idx; i < N; i += stride) { // Process multiple elements per thread for better occupancy
        X[i * INCX] *= ALPHA;
    }
}