#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float temp_mean = 0.0f; // Temporary variable for each thread
    int filter_offset = i * spatial; // Precompute offset for filter

    for (int j = 0; j < batch; ++j) {
        int batch_offset = j * filters * spatial; // Precompute offset for each batch
        for (int k = 0; k < spatial; ++k) {
            int index = batch_offset + filter_offset + k;
            temp_mean += x[index];
        }
    }
    mean[i] = temp_mean * scale; // Scale after the loop
}