#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified global index calculation
    int gridSize = gridDim.x * blockDim.x; // Calculate total thread grid size for striding

    while (i < (NX + NY) * B) { // Use loop to cover entire range
        int b = i / (NX + NY);
        int j = i % (NX + NY);

        // Conditional selection and assignment
        OUT[i] = (j < NX) ? X[b * NX + j] : Y[b * NY + j - NX];

        i += gridSize; // Increment by grid size for strided access
    }
}