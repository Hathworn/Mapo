#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    // Calculate global index based on grid and block
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Stride for handling larger sets

    for (int idx = index; idx < n * size * batch; idx += stride) {
        int i = idx % size;
        idx /= size;
        int j = idx % n;
        idx /= n;
        int k = idx;

        output[(k * n + j) * size + i] += biases[j]; // Accumulate bias
    }
}