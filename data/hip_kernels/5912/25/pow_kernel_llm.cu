#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Single-dimensional index calculation
    int stride = gridDim.x * blockDim.x; // Calculate stride for loop unrolling
    
    for (; i < N; i += stride) { // Loop with stride to handle more elements per thread
        Y[i * INCY] = powf(X[i * INCX], ALPHA); // Use powf for single precision optimization
    }
}