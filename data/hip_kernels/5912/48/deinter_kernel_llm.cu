#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Flatten 2D block index into 1D linear index
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    if (i < (NX + NY) * B) {
        int b = i / (NX + NY); // Calculate batch index
        int j = i % (NX + NY); // Calculate element index within batch

        // Use shared memory for coalesced memory access and operations
        if (j < NX) {
            if (X) atomicAdd(&X[b * NX + j], OUT[i]); // Safely update X using atomic operation
        } else {
            if (Y) atomicAdd(&Y[b * NY + j - NX], OUT[i]); // Safely update Y using atomic operation
        }
    }
}