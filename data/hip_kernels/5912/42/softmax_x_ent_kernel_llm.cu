#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation for 1D grid
    if(i < n){
        float t = truth[i];
        float p = pred[i];
        error[i] = (t > 0.0f) ? -logf(p) : 0.0f;  // Use logf for potential slight performance gain
        delta[i] = t - p;
    }
}