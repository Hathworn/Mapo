#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateExCovX(double *e_x_cov_x, double gauss_d2, int valid_voxel_num)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    // Parallelize using loop unrolling for efficiency
    for (int i = id; i < valid_voxel_num; i += stride * 4) {
        e_x_cov_x[i] *= gauss_d2;
        if (i + stride < valid_voxel_num)
            e_x_cov_x[i + stride] *= gauss_d2;
        if (i + stride * 2 < valid_voxel_num)
            e_x_cov_x[i + stride * 2] *= gauss_d2;
        if (i + stride * 3 < valid_voxel_num)
            e_x_cov_x[i + stride * 3] *= gauss_d2;
    }
}