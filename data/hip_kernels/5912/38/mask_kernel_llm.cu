#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (i < n && mask[i] == mask_num)              // Combined condition on a single line
    {
        x[i] = val;                                // Indented consistently
    }
}