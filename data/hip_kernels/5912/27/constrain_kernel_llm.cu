#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Combine blockIdx calculations into one variable
    int stride = blockDim.x * gridDim.x;               // Calculate full stride across the grid
    for(int i = index; i < N; i += stride) {           // Use loop to process multiple elements per thread
        X[i*INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i*INCX])); // Apply the constraint
    }
}