#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float variance_i = variance[i];
    float mean_i = mean[i];
    float sum = 0.0f;

    // Precompute constant for variance_delta scaling
    float scale = -.5f * powf(variance_i + .00001f, -1.5f);

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += delta[index] * (x[index] - mean_i);
        }
    }

    variance_delta[i] = sum * scale;
}