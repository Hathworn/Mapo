#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Updated to use 1D grid for better efficiency
    if (index >= N) return;

    int f = (index / spatial) % filters;

    // Pre-compute constant terms for reduced calculations
    float inv_std_dev = 1.f / sqrtf(variance[f] + .00001f);
    float spatial_batch_reciprocal = 1.f / (spatial * batch);
    float delta_value = delta[index];
    float x_value = x[index];
    
    delta[index] = delta_value * inv_std_dev
                 + variance_delta[f] * 2.f * (x_value - mean[f]) * spatial_batch_reciprocal
                 + mean_delta[f] * spatial_batch_reciprocal;
}