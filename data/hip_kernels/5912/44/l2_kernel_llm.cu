#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Ensure that our thread processes within bounds
    if (i < n) {
        float diff = truth[i] - pred[i];
        error[i] = diff * diff; // Compute squared error
        delta[i] = diff;  // Store simple difference
    }
}