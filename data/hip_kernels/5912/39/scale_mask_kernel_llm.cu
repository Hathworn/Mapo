#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate 1D index within the grid
    int stride = gridDim.x * blockDim.x;            // Calculate grid stride

    for (; i < n; i += stride) {                    // Loop over elements with grid stride
        if (mask[i] == mask_num)                    // Conditional check for mask number
            x[i] *= scale;                          // Scale the element if condition is true
    }
}