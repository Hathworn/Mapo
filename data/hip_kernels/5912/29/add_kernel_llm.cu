#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Compute 1D thread index
    int stride = blockDim.x * gridDim.x; // Calculate stride for grid
    for (int idx = i; idx < N; idx += stride) // Loop over grid-stride
    {
        X[idx * INCX] += ALPHA; // Perform computation
    }
}