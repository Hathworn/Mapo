#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Precompute scaling factor outside of loops
    float scale = 1.f / (batch * spatial - 1);
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float sum = 0;  // Accumulator variable
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            float diff = x[index] - mean[i];  // Compute difference once
            sum += diff * diff;  // Accumulate squared difference
        }
    }
    variance[i] = sum * scale;  // Scale the accumulated result
}