#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D block indexing
    int stride = blockDim.x * gridDim.x; // Calculate the stride

    // Loop over the data with a stride
    for (int idx = i; idx < N; idx += stride) {
        Y[idx * INCY] *= X[idx * INCX]; // Optimize memory access pattern
    }
}