#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop to handle case where N is larger than total threads available
    for (; i < N; i += blockDim.x * gridDim.x) {
        // Calculate output indices
        int out_index = i;
        int out_w = i % (w * stride);
        i /= (w * stride);
        int out_h = i % (h * stride);
        i /= (h * stride);
        int out_c = i % c;
        i /= c;
        int b = i % batch;

        // Calculate corresponding input indices
        int in_w = out_w / stride;
        int in_h = out_h / stride;
        int in_c = out_c;

        int in_index = b * w * h * c + in_c * w * h + in_h * w + in_w;

        // Perform upsample
        if (forward) {
            out[out_index] += scale * x[in_index];
        } else {
            atomicAdd(x + in_index, scale * out[out_index]);
        }
    }
}