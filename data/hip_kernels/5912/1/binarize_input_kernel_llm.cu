#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = blockIdx.x * blockDim.x + threadIdx.x;
    if (s >= size) return;

    float mean = 0.0f;

    // Use loop unrolling for partial vectorization
    for(int i = 0; i < n; i += 4) {
        if(i < n) mean += fabsf(input[i * size + s]);
        if(i + 1 < n) mean += fabsf(input[(i + 1) * size + s]);
        if(i + 2 < n) mean += fabsf(input[(i + 2) * size + s]);
        if(i + 3 < n) mean += fabsf(input[(i + 3) * size + s]);
    }
    
    mean /= n;

    for(int i = 0; i < n; i += 4) {
        if(i < n) binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
        if(i + 1 < n) binary[(i + 1) * size + s] = (input[(i + 1) * size + s] > 0) ? mean : -mean;
        if(i + 2 < n) binary[(i + 2) * size + s] = (input[(i + 2) * size + s] > 0) ? mean : -mean;
        if(i + 3 < n) binary[(i + 3) * size + s] = (input[(i + 3) * size + s] > 0) ? mean : -mean;
    }
}