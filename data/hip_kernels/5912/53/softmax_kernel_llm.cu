#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    extern __shared__ float sharedMemory[]; // Allocate shared memory
    float *sharedInput = sharedMemory; // Input in shared memory
    float *sharedOutput = sharedMemory + n; // Output in shared memory
    float sum = 0;
    float largest = -INFINITY;

    // Load input into shared memory
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        sharedInput[i] = input[i * stride];
    }
    __syncthreads(); // Synchronize threads

    // Compute the largest value in parallel
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        atomicMax((int *)&largest, __float_as_int(sharedInput[i]));
    }
    __syncthreads(); // Ensure largest is updated

    // Calculate exponential values
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        sharedOutput[i] = expf(sharedInput[i] / temp - largest / temp);
        atomicAdd(&sum, sharedOutput[i]);
    }
    __syncthreads(); // Ensure sum is updated

    // Normalize output
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        sharedOutput[i] /= sum;
        output[i * stride] = sharedOutput[i];
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= batch * groups) return;
    int b = id / groups;
    int g = id % groups;
    extern __shared__ float sharedMemory[]; // Shared memory for this kernel
    softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}