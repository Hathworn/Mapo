#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use grid stride loop to ensure all threads are utilized
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        // Minimize repeated memory access by using a local variable
        float value = X[i * INCX];
        if((value * value) < (ALPHA * ALPHA)) {
            X[i * INCX] = 0;
        }
    }
}