#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    // Calculate spatial dimensions once per thread block
    extern __shared__ int shared_mem[];
    int *sh_w = shared_mem;
    int *sh_h = sh_w + 1;

    int tx = threadIdx.x;
    if (tx == 0) {
        *sh_w = (in_w + 2 * pad) / stride;
        *sh_h = (in_h + 2 * pad) / stride;
    }
    __syncthreads();

    int c = in_c;
    int area = (size-1)/stride;

    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int index = id;
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    int w_offset = -pad;
    int h_offset = -pad;

    float d = 0.0f;

    // Consolidate valid check calculations
    #pragma unroll
    for (int l = -area; l <= area; ++l) {
        for (int m = -area; m <= area; ++m) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;
            bool valid = out_w >= 0 && out_h >= 0 && out_w < *sh_w && out_h < *sh_h;
            if (valid) {
                int out_index = out_w + (*sh_w) * (out_h + (*sh_h) * (k + c * b));
                if (indexes[out_index] == index) {
                    d += delta[out_index];
                }
            }
        }
    }
    prev_delta[index] += d;
}