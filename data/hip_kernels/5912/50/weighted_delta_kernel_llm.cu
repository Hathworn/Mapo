#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Use shared memory if beneficial on target hardware

    if (i < n) {
        float s_val = s[i];
        float dc_val = dc[i];
        if (da) da[i] += dc_val * s_val;  // Use temporary variable to reduce memory access
        if (db) db[i] += dc_val * (1 - s_val);
        ds[i] += dc_val * (a[i] - b[i]);
    }
}