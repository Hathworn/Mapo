#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D grid for simple linear indexing
    int stride = blockDim.x * gridDim.x; // Calculate stride for 1D grid
    for (; i < N; i += stride) { // Loop with stride to process more elements
        X[i * INCX] = ALPHA;
    }
}