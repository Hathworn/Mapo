#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;

    int b = index / spatial;
    int i = index % spatial;
    int baseIndex = b * filters * spatial + i; // Pre-compute base index for efficiency
    float sum = 0.0f;

    // Unrolling loop for efficiency
    for (int f = 0; f < filters; ++f)
    {
        int curIndex = baseIndex + f * spatial; // Use pre-computed base index
        float val = x[curIndex];
        sum += val * val; // Use multiplication instead of powf
    }
    sum = sqrtf(sum);
    sum = (sum == 0.0f) ? 1.0f : sum; // Conditional operator instead of if

    for (int f = 0; f < filters; ++f)
    {
        int curIndex = baseIndex + f * spatial;
        x[curIndex] /= sum; // Normalize x
        dx[curIndex] = (1.0f - x[curIndex]) / sum; // Compute dx
    }
}
```
