#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access by reducing index calculations
__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;
    int size_n = size * n; // Precompute size * n for reuse
    int b, i;
    
    // Use register to optimize repeated index calculations
    for (b = 0; b < batch; ++b) {
        int base_index1 = b * size_n + f1 * size;
        int base_index2 = b * size_n + f2 * size;
        
        for (i = 0; i < size; ++i) {
            int i1 = base_index1 + i;
            int i2 = base_index2 + i;
            sum += output[i1] * output[i2];
            norm1 += output[i1] * output[i1];
            norm2 += output[i2] * output[i2];
        }
    }
    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    sum = sum / norm;
    
    for (b = 0; b < batch; ++b) {
        int base_index1 = b * size_n + f1 * size;
        int base_index2 = b * size_n + f2 * size;
        
        for (i = 0; i < size; ++i) {
            int i1 = base_index1 + i;
            int i2 = base_index2 + i;
            atomicAdd(&delta[i1], -scale * sum * output[i2] / norm);
            atomicAdd(&delta[i2], -scale * sum * output[i1] / norm);
        }
    }
}