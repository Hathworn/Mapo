#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate the global index with improved readability
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Check if index is within bounds before operating
    if (i < n) {
        // Perform multiply-add operation with a single memory access for c[i]
        float temp = a[i] * b[i];
        atomicAdd(&c[i], temp); // Ensure thread-safety with atomic operation
    }
}