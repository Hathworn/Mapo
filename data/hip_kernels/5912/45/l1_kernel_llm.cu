#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Parallelize condition check using unified thread calculation
    if(i < n){
        float diff = truth[i] - pred[i];
        
        // Directly assign abs value using fabsf for floats
        error[i] = fabsf(diff);
        
        // Use ternary operator for concise conditional logic
        delta[i] = (diff > 0.0f) ? 1.0f : -1.0f;
    }
}