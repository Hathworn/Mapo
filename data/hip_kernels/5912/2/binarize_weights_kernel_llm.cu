#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Use shared memory to reduce global memory access
    extern __shared__ float shm[];
    float *shared_weights = shm;

    // Load weights into shared memory
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[i] = fabsf(weights[f * size + i]);
    }
    __syncthreads();

    // Calculate mean in shared memory
    float mean = 0;
    for (int i = 0; i < size; ++i) {
        mean += shared_weights[i];
    }
    mean /= size;

    // Binarize weights based on calculated mean
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    }
}