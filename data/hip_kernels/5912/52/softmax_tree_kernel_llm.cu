#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;

    // Find the largest value
    for (i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = (val > largest) ? val : largest;
    }

    // Compute exponentials and their sum
    for (i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp);
        sum += e;
        output[i * stride] = e;
    }

    // Normalize the output
    float inv_sum = 1.0f / sum;
    for (i = 0; i < n; ++i) {
        output[i * stride] *= inv_sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;

    // Calculate indices
    int s = id % spatial;
    int remainder = id / spatial;
    int g = remainder % groups;
    int b = remainder / groups;

    // Offset calculations
    int goff = group_offset[g] * spatial;
    int boff = b * stride;

    // Perform softmax computation
    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}