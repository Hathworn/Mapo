#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Calculate global thread ID
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= N) return;

    // Calculate input coordinates
    int in_w = index % w;
    int in_h = (index / w) % h;
    int in_c = (index / (w * h)) % c;
    int b = index / (w * h * c);

    // Calculate output channel
    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;

    // Calculate output width and height
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;

    // Compute output index
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    // Perform the reorganization
    if(forward)
        out[out_index] = x[index];
    else
        out[index] = x[out_index];
}