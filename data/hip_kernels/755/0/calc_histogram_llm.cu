#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calc_histogram(char* dbuff, unsigned int* dcount, unsigned int size, float stride) {

    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int start_pos = stride * index;
    unsigned int stop_pos = min(start_pos + stride, size); // Use min to avoid overflow
    unsigned int lcount[10] = { 0 };

    for (unsigned int i = start_pos; i < stop_pos; i++) {
        // Use a switch-case to efficiently increment the counter
        switch (dbuff[i]) {
            case '0': lcount[0]++; break;
            case '1': lcount[1]++; break;
            case '2': lcount[2]++; break;
            case '3': lcount[3]++; break;
            case '4': lcount[4]++; break;
            case '5': lcount[5]++; break;
            case '6': lcount[6]++; break;
            case '7': lcount[7]++; break;
            case '8': lcount[8]++; break;
            case '9': lcount[9]++; break;
        }
    }

    // Use atomicAdd to safely update global counter with local count
    atomicAdd(&dcount[0], lcount[0]);
    atomicAdd(&dcount[1], lcount[1]);
    atomicAdd(&dcount[2], lcount[2]);
    atomicAdd(&dcount[3], lcount[3]);
    atomicAdd(&dcount[4], lcount[4]);
    atomicAdd(&dcount[5], lcount[5]);
    atomicAdd(&dcount[6], lcount[6]);
    atomicAdd(&dcount[7], lcount[7]);
    atomicAdd(&dcount[8], lcount[8]);
    atomicAdd(&dcount[9], lcount[9]);
}