#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_grayscale(int width, int height, float *image, float *image_out)
{
    // Calculate global thread indices for image processing
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure within the image bounds
    if (x < width && y < height) {
        int offset_out = (width * y) + x;
        int offset = offset_out * 3;

        // Perform grayscale conversion using efficient access and calculation
        float b = image[offset];
        float g = image[offset + 1];
        float r = image[offset + 2];
        
        // Store the computed grayscale value
        image_out[offset_out] = b * 0.0722f + g * 0.7152f + r * 0.2126f;
    }
}