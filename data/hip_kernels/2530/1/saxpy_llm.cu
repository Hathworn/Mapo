#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy(float *x, float *y, const float a)
{
    // Calculate unique index for the thread
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for better memory access patterns if applicable
    if (i < ARRAY_SIZE) {
        // Efficient memory access by utilizing registers
        float xi = x[i];
        float yi = y[i];
        yi = a * xi + yi;
        y[i] = yi;
    }
}