#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __host__ float cpu_applyFilter(float *image, int stride, float *matrix, int filter_dim)
{
    float pixel = 0.0f;

    for (int h = 0; h < filter_dim; h++)
    {
        int offset = h * stride;
        int offset_kernel = h * filter_dim;

        for (int w = 0; w < filter_dim; w++)
        {
            pixel += image[offset + w] * matrix[offset_kernel + w];
        }
    }

    return pixel;
}

__global__ void gpu_sobel(int width, int height, float *image, float *image_out)
{
    __shared__ float sh_block[BLOCK_SIZE_SH * BLOCK_SIZE_SH];
    
    float sobel_x[9] = { 1.0f,  0.0f, -1.0f,
                         2.0f,  0.0f, -2.0f,
                         1.0f,  0.0f, -1.0f };
    float sobel_y[9] = { 1.0f,  2.0f,  1.0f,
                         0.0f,  0.0f,  0.0f,
                        -1.0f, -2.0f, -1.0f };

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    int offset_t = index_y * width + index_x; // Input for function
    int offset = (index_y + 1) * width + (index_x + 1); // Output to store in result

    // Shared memory offset (for input value):
    int offset_shared = threadIdx.y * BLOCK_SIZE_SH + threadIdx.x;

    // Load shared memory with boundary checks
    if (index_x < width && index_y < height) {
        sh_block[offset_shared] = image[offset_t];
        
        // Load extra elements for edges
        if (threadIdx.x < 2 && (index_x + 2) < width) {
            sh_block[offset_shared + 1] = image[offset_t + 1];
            sh_block[offset_shared + 2] = image[offset_t + 2];
        }
        if (threadIdx.y < 2 && (index_y + 2) < height) {
            sh_block[offset_shared + BLOCK_SIZE_SH] = image[offset_t + width];
            sh_block[offset_shared + 2 * BLOCK_SIZE_SH] = image[offset_t + 2 * width];
        }
        if (threadIdx.x < 2 && threadIdx.y < 2 && (index_x + 2) < width && (index_y + 2) < height) {
            sh_block[offset_shared + BLOCK_SIZE_SH + 1] = image[offset_t + width + 1];
            sh_block[offset_shared + BLOCK_SIZE_SH + 2] = image[offset_t + width + 2];
            sh_block[offset_shared + 2 * BLOCK_SIZE_SH + 1] = image[offset_t + 2 * width + 1];
            sh_block[offset_shared + 2 * BLOCK_SIZE_SH + 2] = image[offset_t + 2 * width + 2];
        }
    }
    __syncthreads();

    if (index_x < (width - 2) && index_y < (height - 2))
    {
        float gx = cpu_applyFilter(&sh_block[offset_shared], BLOCK_SIZE_SH, sobel_x, 3);
        float gy = cpu_applyFilter(&sh_block[offset_shared], BLOCK_SIZE_SH, sobel_y, 3);
        image_out[offset] = sqrtf(gx * gx + gy * gy);
    }
}