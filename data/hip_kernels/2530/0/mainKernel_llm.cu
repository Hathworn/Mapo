#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define N 1
#define TPB 256

__global__ void mainKernel()
{
    // Use early return for threads not participating in the work
    if (threadIdx.x >= N) return;

    // Directly use block and thread index for efficient computation
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello world! My threadId is %d\n", threadId);
}