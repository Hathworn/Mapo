#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __host__ float cpu_applyFilter(float *image, int stride, float *matrix, int filter_dim)
{
    float pixel = 0.0f;

    for (int h = 0; h < filter_dim; h++)
    {
        int offset        = h * stride;
        int offset_kernel = h * filter_dim;

        for (int w = 0; w < filter_dim; w++)
        {
            pixel += image[offset + w] * matrix[offset_kernel + w];
        }
    }

    return pixel;
}

__global__ void gpu_gaussian(int width, int height, float *image, float *image_out)
{
    // Pre-compute effective block size:
    const int blockSizeEffective = blockDim.x * blockDim.y;
    
    // Define Gaussian filter:
    __shared__ float gaussian[9];
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        gaussian[0] = 1.0f / 16.0f; gaussian[1] = 2.0f / 16.0f; gaussian[2] = 1.0f / 16.0f;
        gaussian[3] = 2.0f / 16.0f; gaussian[4] = 4.0f / 16.0f; gaussian[5] = 2.0f / 16.0f;
        gaussian[6] = 1.0f / 16.0f; gaussian[7] = 2.0f / 16.0f; gaussian[8] = 1.0f / 16.0f;
    }
    __syncthreads();

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    int offset_t = index_y * width + index_x; // Input for function

    // Prepare input for shared memory:
    __shared__ float sh_block[BLOCK_SIZE_SH * BLOCK_SIZE_SH];
    
    // Load image data into shared memory:
    if (index_x < width && index_y < height) {
        sh_block[threadIdx.y * BLOCK_SIZE_SH + threadIdx.x] = image[offset_t];
    }
    __syncthreads();

    int offset = (index_y + 1) * width + (index_x + 1); // Output to store in result

    // Ensure valid execution within image bounds:
    if (index_x < (width - 2) && index_y < (height - 2))
    {
        image_out[offset] = cpu_applyFilter(&sh_block[threadIdx.y * BLOCK_SIZE_SH + threadIdx.x],
                                            BLOCK_SIZE_SH, gaussian, 3);
    }
}