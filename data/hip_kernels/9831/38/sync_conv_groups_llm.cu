#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel does not require unnecessary computation, keeping as a sync point
__global__ void sync_conv_groups() { 
    // Use __syncthreads() to ensure all threads in the block reach this point before proceeding
    __syncthreads();
}