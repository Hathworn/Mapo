#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Use blockDim.x to ensure we handle all threads in a grid/block
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Ensure we do not exceed the size of the array
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}