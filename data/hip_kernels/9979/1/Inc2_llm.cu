#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
Copyright (c) 2015-2016 Advanced Micro Devices, Inc. All rights reserved.
Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:
The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.
*/

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc2(float *Ad, float *Bd) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if(tx < LEN) {
        float curr_val = Ad[tx];
        #pragma unroll 4 // Unroll inner loop for potential performance gain
        for(int i = 0; i < ITER; i++) {
            curr_val += 1.0f;
            Bd[tx] = curr_val;
        }
        Ad[tx] = curr_val; // Write back updated value
    }
}