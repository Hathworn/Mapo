#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
Copyright (c) 2015-2016 Advanced Micro Devices, Inc. All rights reserved.
Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:
The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.
THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN
THE SOFTWARE.
*/

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc1(float *Ad, float *Bd) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure thread index is within bounds
    if(tx < 1) {
        float a_val = Ad[tx]; // Cache value of Ad[tx] to reduce global memory accesses

        for(int i = 0; i < ITER; i++) {
            a_val += 1.0f; // Increment cached value instead of accessing global memory

            // Copy updated value to Bd[tx]
            for(int j = 0; j < 256; j++) {
                Bd[tx] = a_val;
            }
        }

        // Update Ad[tx] with final increment result
        Ad[tx] = a_val;
    }
}