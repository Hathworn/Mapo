#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void HelloFromGPU(void)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    
    if (idx == 0) // Ensure message is printed only once
    {
        printf("Hello World From GPU!\n");
    }
}