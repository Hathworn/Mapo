#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void hellofromGPU(void)
{
    // Each thread prints a message, optimize by using only one thread
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        printf("hello world  \n");
    }
}