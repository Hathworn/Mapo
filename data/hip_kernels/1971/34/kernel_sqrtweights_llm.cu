#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_sqrtweights(int N, double *wt) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for better data caching
    extern __shared__ double shared_wt[];
    
    // Load data into shared memory
    if (tid < N) {
        shared_wt[threadIdx.x] = wt[tid];
        __syncthreads(); // Wait for all threads to load data

        // Compute square root and store back in shared memory
        shared_wt[threadIdx.x] = sqrt(shared_wt[threadIdx.x]);
        __syncthreads(); // Wait for all threads to complete computation

        // Store result back to global memory
        wt[tid] = shared_wt[threadIdx.x];
    }
}