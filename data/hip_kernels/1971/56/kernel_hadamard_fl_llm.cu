#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_hadamard_fl(int N, float *wt, float *x) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop unrolling to handle multiple elements in a single thread if possible
    for (int i = tid; i < N; i += gridDim.x * blockDim.x) {
        x[i] *= wt[i];
    }
}