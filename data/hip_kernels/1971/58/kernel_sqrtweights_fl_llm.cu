#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_sqrtweights_fl(int N, float *wt) {
    // Calculate global thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Process multiple elements per thread for better efficiency
    for (unsigned int i = tid; i < N; i += blockDim.x * gridDim.x) {
        wt[i] = sqrtf(wt[i]);
    }
}