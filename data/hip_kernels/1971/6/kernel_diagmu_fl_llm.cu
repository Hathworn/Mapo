#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_diagmu_fl(int M, float *A, float mu) {
    // Calculate linear index for the thread
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure tid is within the bounds
    if (tid < M) {
        int index = tid * (M + 1); // Calculate the index into A
        A[index] += mu; // Update value with mu
    }
}