#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_setweights_fl(int N, float *wt, float alpha) {
    // Calculate the global thread index
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure only valid threads proceed
    if (tid < N) {
        // Directly set the weight to alpha
        wt[tid] = alpha;
    }
}