#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double digamma(double x) {
    double result = 0.0, xx, xx2, xx4;
    for ( ; x < 7.0; ++x) { // Reduce x till x < 7
        result -= 1.0/x;
    }
    x -= 1.0/2.0;
    xx = 1.0/x;
    xx2 = xx*xx;
    xx4 = xx2*xx2;
    result += log(x)+(1./24.)*xx2-(7.0/960.0)*xx4+(31.0/8064.0)*xx4*xx2-(127.0/30720.0)*xx4*xx4;
    return result;
}

__global__ void kernel_evaluatenu(int Nd, double qsum, double *q, double deltanu, double nulow) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < Nd) {
        double thisnu = (nulow + ((double)tid) * deltanu);
        double dgm1 = digamma(thisnu * 0.5 + 0.5); // Precompute digamma values
        double dgm2 = digamma(thisnu * 0.5);
        
        // Simplified assignments and calculations
        double log_term1 = log((thisnu + 1.0) * 0.5);
        double log_term2 = log(thisnu * 0.5);
        
        q[tid] = dgm1 - log_term1 - dgm2 + log_term2 - qsum + 1.0; // Consolidate calculations
    }
}