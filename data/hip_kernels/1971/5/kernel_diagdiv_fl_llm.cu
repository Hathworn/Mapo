#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_diagdiv_fl(int M, float eps, float *y, float *x) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure this thread processes only relevant data
    if (tid < M) {
        float x_val = x[tid];  // Cache x[tid] to reduce memory access
        if (x_val > eps) {
            y[tid] /= x_val;  // Perform division operation in place
        } else {
            y[tid] = 0.0f;
        }
    }
}