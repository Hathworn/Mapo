#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_hadamard(int N, double *wt, double *x) {
    // Calculate the global thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread ID is within bounds and align memory access
    if (tid < N) {
        x[tid] *= wt[tid];
    }
}