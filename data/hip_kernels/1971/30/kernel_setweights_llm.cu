#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_setweights(int N, double *wt, double alpha) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a stride loop to allow more efficient utilization of threads
    for (unsigned int i = tid; i < N; i += gridDim.x * blockDim.x) {
        wt[i] = alpha;
    }
}