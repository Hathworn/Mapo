#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_updateweights_fl(int N, float *wt, float *x, float *q, float nu){
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if thread is within bounds
    if (tid < N) {
        float x_val = x[tid];
        float wt_val = (nu + 1.0f) / (nu + x_val * x_val); // Calculate weight
        wt[tid] = wt_val;
        q[tid] = wt_val - __logf(wt_val); // Use fast math intrinsic for logarithm
    }
}