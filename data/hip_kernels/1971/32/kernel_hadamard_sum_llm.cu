#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_hadamard_sum(int N, double *y, double *x, double *w) {
    // Use const memory for block size and grid size to optimize performance.
    const unsigned int blockSize = blockDim.x;
    const unsigned int gridSize = gridDim.x * blockSize;
    unsigned int tid = blockIdx.x * blockSize + threadIdx.x;

    // Unrolling loop for better memory access pattern.
    for (int i = tid; i < N; i += gridSize) {
        y[i] += x[i] * w[i];
    }
}