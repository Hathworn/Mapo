#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread index to ensure that only one thread updates accuracy
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}