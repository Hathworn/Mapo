#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bestFilter(const double *Params, const float *data, const float *mu, const float *lam, const float *nu, float *xbest, float *err, int *ftype) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int NT = (int) Params[0];
    int Nfilt = (int) Params[1];
    float Th = (float) Params[2];
    float epu = (float) Params[8];

    int tid0 = tid + bid * blockDim.x; // Use blockDim.x instead of Nthreads
    if (tid0 < NT - 1 && tid0 > 0) { // Use logical AND (&&) for better clarity
        float Cbest = 0.0f, xb = 0.0f;
        int ibest = 0;
        
        for (int i = 0; i < Nfilt; i++) {
            float Ci = data[tid0 + NT * i] + mu[i] * lam[i];
            float Cf = Ci * Ci / (lam[i] + 1.0f) - lam[i] * mu[i] * mu[i];
            
            // Add the shift component
            float cdiff = data[tid0 + 1 + NT * i] - data[tid0 - 1 + NT * i];
            Cf += cdiff * cdiff / (epu + nu[i]);
            
            if (Cf > Cbest) {
                Cbest = Cf;
                xb = Ci - mu[i] * lam[i]; // Simplify the division
                ibest = i;
            }
        }
        
        if (Cbest > Th * Th) {
            err[tid0] = Cbest;
            xbest[tid0] = xb;
            ftype[tid0] = ibest;
        }
    }
}