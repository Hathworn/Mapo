#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// create an image buffer.  return host ptr, pass out device pointer through pointer to pointer
__global__ void resultant(unsigned char *a, unsigned char *b, unsigned char *c)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Load data into registers to reduce global memory access
    float opposite_side = float(a[idx]);
    float adjacent_side = float(b[idx]);

    // Use fused multiply-add (FMA) to improve performance
    float sum = opposite_side + adjacent_side;
    unsigned char result = (unsigned char)sqrtf(fmaf(sum, sum, -2 * opposite_side * adjacent_side));

    // Apply condition
    if (result > 15 && (idx == 0 || c[idx - 1] != 148))  // Check idx to avoid accessing out-of-bounds
        result = 148;
    else
        result = 0;

    // Store result back to global memory
    c[idx] = result;
}