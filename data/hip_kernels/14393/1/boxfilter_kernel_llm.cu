#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// create an image buffer. return host ptr, pass out device pointer through pointer to pointer
__global__ void boxfilter_kernel(int iw, int ih, unsigned char *source, unsigned char *dest, int bw, int bh)
{
    // Calculate our pixel's location
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Check if the pixel is inside the image boundaries
    if (x < iw && y < ih) 
    {
        // Variables to store the sum
        int count = 0;
        float sum = 0.0f;

        // Do the blur operation by summing the surround pixels
        for (int j = -(bh / 2); j <= (bh / 2); j++)
        {
            for (int i = -(bw / 2); i <= (bw / 2); i++)
            {
                // Verify that this offset is within the image boundaries
                int xi = x + i;
                int yj = y + j;
                if (xi < iw && xi >= 0 && yj < ih && yj >= 0)
                {
                    sum += (float) source[(yj * iw) + xi];
                    count++;
                }
            }
        }

        // Average the sum
        sum /= (float) count;
        dest[(y * iw) + x] = (unsigned char) sum;
    }
}