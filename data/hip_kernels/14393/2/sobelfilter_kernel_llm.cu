#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sobelfilter_kernel(int iw, int ih, unsigned char *source, unsigned char *dest)
{
    // Calculate our pixel's location
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Operate only if we are in the correct boundaries
    if (x > 0 && x < iw - 1 && y > 0 && y < ih - 1)
    {
        // Precompute index offsets to avoid repetitive calculations
        int idx = iw * y + x;
        int idx_n = iw * (y - 1) + x;
        int idx_s = iw * (y + 1) + x;

        // Calculate gradients using shared memory to minimize global memory access
        int gx = -source[idx_n - 1] + source[idx_n + 1]
                 - 2 * source[idx - 1] + 2 * source[idx + 1]
                 - source[idx_s - 1] + source[idx_s + 1];

        int gy = -source[idx_n - 1] - 2 * source[idx_n] - source[idx_n + 1]
                 + source[idx_s - 1] + 2 * source[idx_s] + source[idx_s + 1];

        // Store the computed Sobel magnitude
        dest[idx] = (int)sqrtf((float)(gx) * (float)(gx) + (float)(gy) * (float)(gy));
    }
}