#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load data into register for faster access
    float acc_val = *accuracy;
    
    // Perform division
    acc_val /= N;

    // Store result back to global memory
    *accuracy = acc_val;
}