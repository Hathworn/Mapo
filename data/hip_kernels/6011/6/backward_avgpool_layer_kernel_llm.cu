#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n * w * h) return;  // Simplify boundary condition

    int pool_size = w * h;  // Pre-calculate pool size
    int b = id / pool_size; // Compute batch index
    int i = id % pool_size; // Compute local pool index
    
    int k = b % c;          // Compute channel index
    b /= c;

    int in_index = i + pool_size * (k + b * c);
    int out_index = k + c * b;

    in_delta[in_index] += out_delta[out_index] / pool_size;  // Use pre-calculated pool_size
}