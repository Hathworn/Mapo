#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    // Optimize index calculation by removing repeated operations
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if(id < size) {
        // Use conditional operator directly in assignment
        input[id] *= (rand[id] >= prob) ? scale : 0;
    }
}