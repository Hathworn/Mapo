#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device_new_api(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    for (i = 0; i < n; ++i) {
        float val = input[i * stride]; // Optimize data type from int to float
        largest = fmaxf(val, largest); // Using intrinsic for max
    }
    for (i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp); // Precompute part of the exp
        sum += e;
        output[i * stride] = e;
    }
    sum = 1.0f / sum; // Compute reciprocal once
    for (i = 0; i < n; ++i) {
        output[i * stride] *= sum; // Use precomputed reciprocal
    }
}

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    for(i = 0; i < n; ++i){
        float val = input[i]; // Optimize data type from int to float
        largest = fmaxf(val, largest); // Using intrinsic for max
    }
    for(i = 0; i < n; ++i){
        float e = expf((input[i] - largest) / temp); // Precompute part of the exp
        sum += e;
        output[i] = e;
    }
    sum = 1.0f / sum; // Compute reciprocal once
    for(i = 0; i < n; ++i){
        output[i] *= sum; // Use precomputed reciprocal
    }
}

__global__ void softmax_kernel_new_api(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplify thread ID calculation
    if (id >= batch * groups) return;
    int b = id / groups;
    int g = id % groups;
    softmax_device_new_api(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}