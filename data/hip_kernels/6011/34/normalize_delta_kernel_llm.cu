#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use a loop to allow multiple threads to process multiple elements
    for (; index < N; index += stride) {
        int f = (index/spatial) % filters;

        // Perform normalization and update delta
        float inv_sqrt_variance = 1.F / (sqrtf(variance[f]) + .000001f);
        delta[index] = delta[index] * inv_sqrt_variance 
                     + variance_delta[f] * 2.F * (x[index] - mean[f]) / (spatial * batch) 
                     + mean_delta[f] / (spatial * batch);
    }
}