#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Calculate the scale factor once; all threads in a warp compute concurrently.
    float scale = 1.F / (batch * spatial);

    // Calculate a unique thread index for different grids and threads.
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure that each thread processes a valid filter.
    if (i >= filters) return;

    // Initialize mean[i] to zero using atomic operation
    mean[i] = 0.0f;

    // Use one loop to reduce index calculations, j and k are combined.
    for (int batch_spatial = 0; batch_spatial < batch * spatial; ++batch_spatial)
    {
        // Compute the index using the current i and batch_spatial
        int index = (batch_spatial / spatial) * filters * spatial + i * spatial + (batch_spatial % spatial);
        mean[i] += x[index];
    }

    // Scale the mean value after accumulation to improve numerical stability.
    mean[i] *= scale;
}