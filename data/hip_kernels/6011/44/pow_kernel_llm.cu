#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D grid
    int stride = gridDim.x * blockDim.x; // Calculate stride for more efficient iteration
    for (int idx = i; idx < N; idx += stride) // Loop to cover full range
    {
        Y[idx * INCY] = powf(X[idx * INCX], ALPHA);
    }
}