#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = j * gridDim.x * blockDim.x + i;

    // Coalesced memory access and leveraging more parallel threads
    if(index < N) 
    {
        Y[index * INCY + OFFY] = X[index * INCX + OFFX];
    }
}