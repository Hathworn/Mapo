#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride_w = w * stride;
    size_t stride_h = h * stride;
    if (i >= N) return;
    
    // Precompute repeated values to reduce calculations
    int out_c = (i / (stride_w * stride_h)) % c;
    int b = (i / (stride_w * stride_h * c)) % batch;
    int out_h = (i / stride_w) % stride_h;
    int out_w = i % stride_w;

    // Utilize modulo and division only where necessary for better performance
    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_index = b*w*h*c + out_c*w*h + in_h*w + in_w;

    if (forward) 
        out[i] += scale * x[in_index];
    else 
        atomicAdd(x + in_index, scale * out[i]);
}