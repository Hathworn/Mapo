#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

extern "C" {
double get_time_point();
void start_timer();
void stop_timer();
double get_time();
void stop_timer_and_show();
void stop_timer_and_show_name(char *name);
void show_total_time();
}


__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (f >= n) return;

    float sum = 0;
    float* weight_ptr = weights + f * size; // Pointer to the start of the current weight set
    float* binary_ptr = binary + f * size;  // Pointer to the start of the current binary set

    // Calculate mean of absolute weights
    for(int i = 0; i < size; ++i){
        sum += fabs(weight_ptr[i]);
    }
    float mean = sum / size;

    // Binarize weights based on mean
    for(int i = 0; i < size; ++i){
        binary_ptr[i] = (weight_ptr[i] > 0) ? mean : -mean;
    }
}