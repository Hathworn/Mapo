#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device_new_api(float *input, int n, float temp, int stride, float *output)
{
    float sum = 0;
    float largest = -INFINITY;

    // Parallelize the loop to find the largest element
    for (int i = 0; i < n; ++i) {
        int val = input[i * stride];
        largest = fmaxf(val, largest);
    }
    __syncthreads(); // Ensure all threads have updated the largest value

    // Parallelize the loop to compute exponential values
    for (int i = 0; i < n; ++i) {
        float e = expf(input[i*stride] / temp - largest / temp);
        atomicAdd(&sum, e); // Safely accumulate the sum using atomicAdd
        output[i*stride] = e;
    }
    __syncthreads(); // Ensure all threads have updated the sum

    // Parallelize the loop to normalize the output
    for (int i = 0; i < n; ++i) {
        output[i*stride] /= sum;
    }
}

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    float sum = 0;
    float largest = -INFINITY;

    // Parallelize the loop to find the largest element
    for (int i = 0; i < n; ++i) {
        int val = input[i];
        largest = fmaxf(val, largest);
    }
    __syncthreads(); // Ensure all threads have updated the largest value

    // Parallelize the loop to compute exponential values
    for (int i = 0; i < n; ++i) {
        float e = exp(input[i]/temp - largest/temp);
        atomicAdd(&sum, e); // Safely accumulate the sum using atomicAdd
        output[i] = e;
    }
    __syncthreads(); // Ensure all threads have updated the sum

    // Parallelize the loop to normalize the output
    for (int i = 0; i < n; ++i) {
        output[i] /= sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;

    int s = id % spatial;
    id = id / spatial;
    int g = id % groups;
    int b = id / groups;
    int goff = group_offset[g] * spatial;
    int boff = b * stride;
    softmax_device_new_api(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}