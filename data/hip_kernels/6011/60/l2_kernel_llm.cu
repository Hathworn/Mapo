#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Use shared memory to minimize global memory access latency
    __shared__ float shared_pred[256];
    __shared__ float shared_truth[256];

    int tid = threadIdx.x;
    int global_index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + tid;

    if (global_index < n) {
        // Load data into shared memory
        shared_pred[tid] = pred[global_index];
        shared_truth[tid] = truth[global_index];
        
        __syncthreads(); // Ensure all threads have loaded their data

        float diff = shared_truth[tid] - shared_pred[tid];
        error[global_index] = diff * diff;
        delta[global_index] = diff;
    }
}