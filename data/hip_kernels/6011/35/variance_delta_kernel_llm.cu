#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return; // Early return if out of bounds
    
    float sum = 0.0f; // Accumulate in register for better performance

    // Loop unrolling might not be beneficial due to dynamic batch and spatial 
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += delta[index] * (x[index] - mean[i]);
        }
    }

    variance_delta[i] = sum * (-0.5f * powf(variance[i] + 0.000001f, -1.5f)); // Compute variance delta
}
```
