#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x; // Simplified index calculation for 1D grid
    if(i < N) {
        float x_val = X[OFFX + i * INCX];         // Cache accessed X value
        Y[OFFY + i * INCY] += ALPHA * x_val;      // Use cached value for updating Y
    }
}