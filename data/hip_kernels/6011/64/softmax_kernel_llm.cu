#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(int n, float *input, float temp, float *output) {
    extern __shared__ float shared_mem[]; // Use shared memory to optimize
    float *e_vals = shared_mem;
    int i = threadIdx.x;
    if (i < n) {
        float sum = 0;
        float largest = -INFINITY;

        // Find the largest value
        for (int j = i; j < n; j += blockDim.x) {
            float val = input[j];
            largest = (val > largest) ? val : largest;
        }
        largest = __shfl_sync(0xFFFFFFFF, largest, 0);

        // Calculate exp values, store temporarily in shared memory and calculate sum
        for (int j = i; j < n; j += blockDim.x) {
            float e = expf(input[j] / temp - largest / temp);
            e_vals[j] = e;
            sum += e;
        }
        sum = __shfl_sync(0xFFFFFFFF, sum, 0);

        // Normalize the output values
        if (i < n) {
            for (int j = i; j < n; j += blockDim.x) {
                output[j] = e_vals[j] / sum;
            }
        }
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output) {
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if (b >= batch) return;
    softmax_device(n, input + b * offset, temp, output + b * offset);
}
```
