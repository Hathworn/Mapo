#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mask_kernel_new_api(int n, float *x, float mask_num, float *mask, float val)
{
    // Calculate the global thread index more concisely
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Optimize the condition to reduce warp divergence
    if (i < n) {
        float current_mask = mask[i];
        if (current_mask == mask_num) {
            x[i] = val;
        }
    }
}