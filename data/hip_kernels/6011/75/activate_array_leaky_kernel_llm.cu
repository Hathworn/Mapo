#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void activate_array_leaky_kernel(float *x, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        // Use register memory for improved performance
        float val = x[index];
        x[index] = (val > 0) ? val : val * 0.1f; // Replace division for better optimization
    }
}