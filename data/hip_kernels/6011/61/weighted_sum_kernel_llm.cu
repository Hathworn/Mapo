#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loops if needed or simplify operations (not applicable here)
    if(i < n){
        float s_val = s[i];
        float a_val = a[i];
        // Use ternary operator outside condition to minimize branching
        float b_val = (b != nullptr) ? b[i] : 0;
        c[i] = s_val * a_val + (1 - s_val) * b_val;
    }
}