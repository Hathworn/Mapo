#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

extern "C" {
double get_time_point();
void start_timer();
void stop_timer();
double get_time();
void stop_timer_and_show();
void stop_timer_and_show_name(char *name);
void show_total_time();
}

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;
    
    // Utilize shared memory for intermediate values
    extern __shared__ float shared_mem[];
    float *shared_mean = shared_mem;
    
    float mean = 0.0f;

    // Calculate sum of absolute inputs for this block
    for (int i = 0; i < n; ++i) {
        mean += fabs(input[i * size + s]);
    }

    // Calculate mean over 'n' and store in shared memory
    shared_mean[threadIdx.x] = mean / n;
    __syncthreads();

    // Assign mean value to binary output, with positive/negative check
    float binarized_value = (input[threadIdx.x * size + s] > 0) ? shared_mean[threadIdx.x] : -shared_mean[threadIdx.x];
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = binarized_value;
    }
}