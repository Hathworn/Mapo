#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified 1D index calculation for efficiency
    if (i < n) {
        float t = truth[i];
        float p = pred[i];
        float log_p = (p > 0) ? log(p) : 0; // Avoid log(0) by checking positivity
        error[i] = (t != 0) ? -log_p : 0; // Only compute log if t is non-zero
        delta[i] = t - p; // Compute the gradient
    }
}