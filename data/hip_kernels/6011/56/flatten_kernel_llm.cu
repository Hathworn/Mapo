#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = blockDim.x * gridDim.x;  // Calculate grid stride

    for (; i < N; i += gridStride) {  // Loop for strided grid access
        int in_s = i % spatial;
        int in_c = (i / spatial) % layers;
        int b = i / (spatial * layers);

        int i1 = b * layers * spatial + in_c * spatial + in_s;
        int i2 = b * layers * spatial + in_s * layers + in_c;

        if (forward) {
            out[i2] = x[i1];
        } else {
            out[i1] = x[i2];
        }
    }
}