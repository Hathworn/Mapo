#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we only process valid elements
    if (i >= N) return;
    
    // Compute the index considering the stride INCX
    int idx = i * INCX;
    
    // Calculate the squared value once for comparison
    float x_squared = X[idx] * X[idx];
    
    // Set to zero if the squared value is less than ALPHA squared
    if (x_squared < (ALPHA * ALPHA)) {
        X[idx] = 0;
    }
}