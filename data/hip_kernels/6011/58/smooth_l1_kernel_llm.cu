#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global index and ensure it doesn't exceed the array length
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff); // Use faster HIP intrinsic for absolute value
        if (abs_val < 1.0f) {
            error[i] = diff * diff;
            delta[i] = diff;
        } else {
            error[i] = 2.0f * abs_val - 1.0f;
            delta[i] = copysignf(1.0f, diff); // Use copysignf for clarity and potential performance gain
        }
    }
}