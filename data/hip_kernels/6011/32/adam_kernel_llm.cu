#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation
    if (index >= N) return;

    float powB1 = 1.F - powf(B1, t); // Precompute repeated calculations
    float powB2 = 1.F - powf(B2, t);
    float sqrt_v = sqrtf(v[index]) + eps;
    
    x[index] -= (rate * sqrtf(powB2) / powB1) * (m[index] / sqrt_v); // Optimize operation order
}