#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolve_gpu_kernel(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n, int size, int pad)
{
    // Calculate unique global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate corresponding x, y, and fil indices
    int x = index % in_w;
    int index2 = index / in_w;
    int y = index2 % in_h;
    int fil = index2 / in_h;

    // Check boundary conditions
    if (fil < n) {

        // Calculate output index
        int const output_index = fil * in_w * in_h + y * in_w + x;
        float sum = 0;

        // Iterate through input channels
        for (int chan = 0; chan < in_c; ++chan) {
            int const weights_pre_index = fil * in_c * size * size + chan * size * size;
            int const input_pre_index = chan * in_w * in_h;

            // Loop unrolling for filter - y and x
            #pragma unroll
            for (int f_y = 0; f_y < size; ++f_y) {
                int input_y = y + f_y - pad;
                #pragma unroll
                for (int f_x = 0; f_x < size; ++f_x) {
                    int input_x = x + f_x - pad;
                    if (input_y < 0 || input_x < 0 || input_y >= in_h || input_x >= in_w) continue;

                    // Calculate input and weights indices
                    int input_index = input_pre_index + input_y * in_w + input_x;
                    int weights_index = weights_pre_index + f_y * size + f_x;

                    // Accumulate weighted sum
                    sum += input[input_index] * weights[weights_index];
                }
            }
        }
        // Store result in output array
        output[output_index] = sum;
    }
}