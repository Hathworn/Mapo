#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    // Compute unique index using threadIdx and blockIdx
    int index = blockIdx.z * n * size + blockIdx.y * size + blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check within bounds and apply bias scaling
    if(index < n * size) output[index] *= biases[blockIdx.y];
}