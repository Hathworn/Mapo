#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Function to convert HSV to RGB
__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    float h = hsv.x;
    float s = hsv.y;
    float v = hsv.z;

    float r, g, b;
    float f, p, q, t;

    // Check saturation
    if (s == 0) {
        r = g = b = v; // Achromatic case
    } else {
        int index = (int) floorf(h); // Calculate index for hue
        f = h - index; // Fractional part of hue
        p = v * (1 - s);
        q = v * (1 - s * f);
        t = v * (1 - s * (1 - f));

        // Use index to select RGB values
        if (index == 0) {
            r = v; g = t; b = p;
        } else if (index == 1) {
            r = q; g = v; b = p;
        } else if (index == 2) {
            r = p; g = v; b = t;
        } else if (index == 3) {
            r = p; g = q; b = v;
        } else if (index == 4) {
            r = t; g = p; b = v;
        } else {
            r = v; g = p; b = q; // Default case
        }
    }

    // Clamp RGB values to [0, 1] range
    r = (r < 0) ? 0 : ((r > 1) ? 1 : r);
    g = (g < 0) ? 0 : ((g > 1) ? 1 : g);
    b = (b < 0) ? 0 : ((b > 1) ? 1 : b);

    return make_float3(r, g, b);
}

// Function to convert RGB to HSV
__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    float r = rgb.x;
    float g = rgb.y;
    float b = rgb.z;

    float h, s, v;
    float max = fmaxf(fmaxf(r, g), b); // Calculate max value
    float min = fminf(fminf(r, g), b); // Calculate min value
    float delta = max - min; // Compute delta
    v = max;

    // Check if max is zero
    if (max == 0) {
        s = 0;
        h = -1;
    } else {
        s = delta / max; // Saturation calculation
        // Hue calculations based on the RGB component with max value
        if (r == max) {
            h = (g - b) / delta;
        } else if (g == max) {
            h = 2 + (b - r) / delta;
        } else {
            h = 4 + (r - g) / delta;
        }
        if (h < 0) h += 6; // Ensure h is positive
    }

    return make_float3(h, s, v);
}

// Optimized global kernel function for image manipulation
__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int size = batch * w * h;
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Compute unique thread ID

    if (id >= size) return; // Out of bounds check

    int x = id % w;
    id /= w;
    int y = id % h;
    id /= h;

    float rshift = rand[0];
    float gshift = rand[1];
    float bshift = rand[2];
    float r0 = rand[8 * id];
    float r1 = rand[8 * id + 1];
    float r2 = rand[8 * id + 2];
    float r3 = rand[8 * id + 3];

    // Adjust saturation based on random values
    saturation = r0 * (saturation - 1) + 1;
    saturation = (r1 > 0.5) ? 1.0 / saturation : saturation;
    // Adjust exposure based on random values
    exposure = r2 * (exposure - 1) + 1;
    exposure = (r3 > 0.5) ? 1.0 / exposure : exposure;

    size_t offset = id * h * w * 3; // Calculate image offset
    image += offset;

    float r = image[x + w * (y + 0 * h)];
    float g = image[x + w * (y + 1 * h)];
    float b = image[x + w * (y + 2 * h)];
    float3 rgb = make_float3(r, g, b);

    // Apply training transformations
    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb); // Convert RGB to HSV
        hsv.y *= saturation; // Adjust saturation
        hsv.z *= exposure; // Adjust exposure
        rgb = hsv_to_rgb_kernel(hsv); // Convert HSV back to RGB
    } else {
        shift = 0;
    }

    // Apply transformations and write back to image
    image[x + w * (y + 0 * h)] = rgb.x * scale + translate + (rshift - 0.5) * shift;
    image[x + w * (y + 1 * h)] = rgb.y * scale + translate + (gshift - 0.5) * shift;
    image[x + w * (y + 2 * h)] = rgb.z * scale + translate + (bshift - 0.5) * shift;
}