#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    // Calculate unique global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Process only valid indices
    if(i < n && __ldg(&mask[i]) == mask_num) 
    {
        x[i] = mask_num;
    }
}