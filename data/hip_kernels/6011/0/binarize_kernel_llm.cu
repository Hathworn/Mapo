#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

extern "C" {
double get_time_point();
void start_timer();
void stop_timer();
double get_time();
void stop_timer_and_show();
void stop_timer_and_show_name(char *name);
void show_total_time();
}

// Optimize kernel by reducing redundant calculations
__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify index calculation
    if (i < n) {                                    // Check within bounds to avoid redundant work
        binary[i] = (x[i] >= 0) ? 1.0f : -1.0f;      // Use floating point constants
    }
}