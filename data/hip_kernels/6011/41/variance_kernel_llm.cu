#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Calculate scale outside of loop as constants.
    float scale = 1.F / (batch * spatial - 1);
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    // Check bounds.
    if (i >= filters) return;

    float local_variance = 0.0f;

    // Loop over batch and spatial dimensions.
    for(int j = 0; j < batch; ++j){
        for(int k = 0; k < spatial; ++k){
            int index = j * filters * spatial + i * spatial + k;
            float diff = x[index] - mean[i]; // Pre-calculate difference.
            local_variance += diff * diff;   // Accumulate squared differences.
        }
    }

    // Apply scale outside the loop.
    variance[i] = local_variance * scale;
}