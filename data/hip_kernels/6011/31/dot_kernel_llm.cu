#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta) 
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int totalElements = batch * n * size;

    // Ensure index falls within valid range
    if (index >= totalElements) return;

    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;

    // Unrolling the loop for better performance
    for (int b = 0; b < batch; ++b) {
        int base_index1 = b * n * size + f1 * size;
        int base_index2 = b * n * size + f2 * size;

        // Vectorized computation
        #pragma unroll
        for (int i = 0; i < size; ++i) {
            int i1 = base_index1 + i;
            int i2 = base_index2 + i;
            float val1 = output[i1];
            float val2 = output[i2];
            sum += val1 * val2;
            norm1 += val1 * val1;
            norm2 += val2 * val2;
        }
    }

    norm1 = rsqrtf(norm1);  // Use reciprocal square root for optimization
    norm2 = rsqrtf(norm2);
    float norm = 1.0f / (norm1 * norm2);
    sum = sum * norm;

    // Unrolling the loop for better performance
    for (int b = 0; b < batch; ++b) {
        int base_index1 = b * n * size + f1 * size;
        int base_index2 = b * n * size + f2 * size;

        #pragma unroll
        for (int i = 0; i < size; ++i) {
            int i1 = base_index1 + i;
            int i2 = base_index2 + i;
            float val1 = output[i1];
            float val2 = output[i2];
            atomicAdd(&delta[i1], -scale * sum * val2 * norm);
            atomicAdd(&delta[i2], -scale * sum * val1 * norm);
        }
    }
}