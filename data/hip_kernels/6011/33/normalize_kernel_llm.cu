#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    // Calculate global index using block index and thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = index; i < N; i += stride) {
        // Compute the filter index
        int f = (i / spatial) % filters;
        
        // Normalize the value
        x[i] = (x[i] - mean[f]) / (sqrtf(variance[f]) + .000001f);
    }
}