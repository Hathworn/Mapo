#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    // Precompute constants for efficiency
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int c = in_c;
    int area = (size - 1) / stride;
    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    // Compute the unique global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Early exit if thread index is out of bounds
    if (id >= n) return;
    
    // Decode index into dimensions
    int index = id;
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    float d = 0;
    
    // Use a single loop with conditional for efficiency
    for (int l = -area; l <= area; ++l) {
        for (int m = -area; m <= area; ++m) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;
            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h) {
                int out_index = out_w + w * (out_h + h * (k + c * b));
                if (indexes[out_index] == index) {
                    d += delta[out_index];
                }
            }
        }
    }
    
    // Accumulate result
    atomicAdd(&prev_delta[index], d);
}