#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void col2im_gpu_kernel(const int n, const float* data_col, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_im) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;

    while (index < n) {
        float val = 0;
        int w = index % width + pad;
        int h = (index / width) % height + pad;
        int c = index / (width * height);

        // Pre-calculate reusable values
        int w_stride = w / stride;
        int h_stride = h / stride;
        int ksize_stride = ksize / stride;

        // Start and end for columns
        int w_col_start = max(0, (w - ksize) / stride + 1);
        int w_col_end = min(w_stride + 1, width_col);
        int h_col_start = max(0, (h - ksize) / stride + 1);
        int h_col_end = min(h_stride + 1, height_col);

        // Offset and coeffs for optimized memory access
        int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col;
        int coeff_h_col = stride * width_col - stride * ksize * height_col * width_col;
        int coeff_w_col = height_col * stride * width_col - stride;

        // Loop through relevant columns in shared memory
        for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
            for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
                val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
            }
        }
        data_im[index] += val;

        index += total_threads; // Advance to the next index for this thread
    }
}