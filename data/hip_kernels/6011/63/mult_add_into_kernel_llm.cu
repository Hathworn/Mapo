#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate unique thread index within grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure valid memory access for array
    if (i < n) {
        // Perform multiplication and addition
        c[i] += a[i] * b[i];
    }
}