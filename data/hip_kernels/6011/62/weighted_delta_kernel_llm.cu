#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        float dc_val = dc[i];
        float s_val = s[i];

        // Minimize repeated global memory reads
        if (da) da[i] += dc_val * s_val;
        db[i] += dc_val * (1.0f - s_val);
        ds[i] += dc_val * (a[i] - b[i]);
    }
}