#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = threadIdx.x + blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x);
    int stride = blockDim.x * gridDim.x * gridDim.y;

    // Loop to handle more data by the same thread
    for (int i = idx; i < N; i += stride) {
        X[i * INCX] = ALPHA;
    }
}