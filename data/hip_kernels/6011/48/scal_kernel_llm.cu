#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;  // Calculate the stride

    // Use a loop to handle larger data sizes more efficiently
    for (; i < N; i += stride) {
        X[i * INCX] *= ALPHA;  // Scale the vector element
    }
}