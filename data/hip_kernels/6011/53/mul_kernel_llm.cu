#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Optimize thread index calculation for 1D grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Use a stride loop to cover all elements
    for(; i < N; i += blockDim.x * gridDim.x)
    {
        Y[i * INCY] *= X[i * INCX];
    }
}