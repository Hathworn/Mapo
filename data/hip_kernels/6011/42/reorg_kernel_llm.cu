#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel optimized for performance
__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return; // Early exit if out of bounds

    int batchSize = w * h * c;
    int in_c = (idx / (w * h)) % c;
    int in_h = (idx / w) % h;
    int in_w = idx % w;

    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * (idx / batchSize))); // Reorganized indexing

    if (forward)
        out[out_index] = x[idx]; // Writing based on forward flag
    else
        out[idx] = x[out_index]; // Reverse write
}