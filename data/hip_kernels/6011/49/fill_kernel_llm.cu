#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread ID using 3D grid and block indexes for better flexibility
    int i = blockIdx.x * blockDim.x + threadIdx.x + 
            blockIdx.y * (blockDim.x * gridDim.x) + 
            blockIdx.z * (blockDim.x * gridDim.x * gridDim.y);

    // Ensure memory accesses are within bounds
    if (i < N) X[i * INCX] = ALPHA;
}