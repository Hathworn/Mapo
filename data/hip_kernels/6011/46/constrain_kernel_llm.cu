#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use a single-dimension grid and calculate global index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N) {
        // Use temporary variable to avoid recalculating index
        int index = i * INCX;
        X[index] = fminf(ALPHA, fmaxf(-ALPHA, X[index]));
    }
}