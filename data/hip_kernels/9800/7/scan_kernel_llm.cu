#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan_kernel(unsigned int* output_block, unsigned int block_num) {
    __shared__ unsigned int shared_output[BLOCK_SIZE];

    // Initialize shared memory using a ternary operator to remove conditional statement
    shared_output[threadIdx.x] = (threadIdx.x >= block_num || threadIdx.x == 0) ? 0x0 : output_block[threadIdx.x - 1];
    __syncthreads();

    // Unroll loops for better performance
    #pragma unroll
    for (unsigned int i = 1; i < block_num; i <<= 1) {
        unsigned int val = (threadIdx.x >= i) ? shared_output[threadIdx.x - i] : 0;
        __syncthreads();
        shared_output[threadIdx.x] += val;
        __syncthreads();
    }

    // Write back the results to global memory
    if (threadIdx.x < block_num) {
        output_block[threadIdx.x] = shared_output[threadIdx.x];
    }
    __syncthreads();
}