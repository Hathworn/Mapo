#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize for shared memory and reduce atomic operations
__global__ void HistogramKernel(unsigned int * input, unsigned int size, unsigned int* histogram, unsigned int pass) {

    extern __shared__ unsigned int localHist[]; // Shared memory for local histogram
    int tid = threadIdx.x;
    int mid = tid + blockIdx.x * blockDim.x;

    // Initialize shared memory
    if (tid < 2) {
        localHist[tid] = 0;
    }
    __syncthreads();

    // Accumulate local histogram
    if (mid < size) {
        atomicAdd(&localHist[(input[mid]>>pass) & 0x01], 1);
    }
    __syncthreads();

    // Reduce local histograms to global histogram
    if (tid < 2) {
        atomicAdd(&histogram[tid], localHist[tid]);
    }
}