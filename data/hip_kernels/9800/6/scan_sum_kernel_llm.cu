#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan_sum_kernel(unsigned int* input_vals, unsigned int pass, unsigned int * output, unsigned int* output_block, unsigned int size, unsigned int block_num) {
    unsigned int tid = threadIdx.x;
    unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ unsigned int shared_input_vals[BLOCK_SIZE];
    __shared__ unsigned int shared_output[BLOCK_SIZE];

    // Load input data to shared memory
    if (mid < size) {
        shared_input_vals[tid] = input_vals[mid];
    } else {
        shared_input_vals[tid] = 0xFFFFFFFF;
    }

    __syncthreads();

    // Compute output values
    if (tid == 0 || ((shared_input_vals[tid - 1] >> pass) & 0x01)) {
        shared_output[tid] = 0;
    } else {
        shared_output[tid] = 1;
    }

    __syncthreads();

    // Perform scan (inclusive prefix sum)
    for (unsigned int i = 1; i < BLOCK_SIZE; i <<= 1) {
        unsigned int val = 0;
        if (tid >= i) {
            val = shared_output[tid - i];
        }
        __syncthreads();
        shared_output[tid] += val;
        __syncthreads();
    }

    // Write results to global memory
    if (mid < size) {
        output[mid] = shared_output[tid];
        // Write block result to output_block
        if (tid == BLOCK_SIZE - 1) {
            output_block[blockIdx.x] = shared_output[tid];
            if (!((shared_input_vals[tid] >> pass) & 0x01)) {
                output_block[blockIdx.x] += 1;
            }
        }
    }
}