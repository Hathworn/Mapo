#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scatter_kernel(unsigned int* d_inputVals, unsigned int* d_inputPos, unsigned int* d_outputVals, unsigned int* d_outputPos, unsigned int* cu_outputVals, size_t numElems) {
    // Compute the global thread index
    unsigned int mid = threadIdx.x + blockIdx.x * blockDim.x;

    if (mid < numElems) {
        // Read the target index from cu_outputVals
        unsigned int val = cu_outputVals[mid];

        // Output the result directly without checking the condition again
        d_outputVals[val] = d_inputVals[mid];
        d_outputPos[val] = d_inputPos[mid];
    }
    // __syncthreads() is unnecessary here and can be removed
}