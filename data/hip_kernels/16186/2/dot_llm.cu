#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(int *a, int *b, int *c) {
    __shared__ int temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int localIndex = threadIdx.x;

    // Load data in shared memory
    temp[localIndex] = a[index] * b[index];
    __syncthreads();

    // Reduce in shared memory tree pattern
    for (int stride = THREADS_PER_BLOCK / 2; stride > 0; stride /= 2) {
        if (localIndex < stride) {
            temp[localIndex] += temp[localIndex + stride];
        }
        __syncthreads();
    }

    // Write result of block to global memory
    if (localIndex == 0) {
        atomicAdd(c, temp[0]);
    }
}
```
