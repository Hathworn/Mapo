#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMult (int *a, int *b, int *c, int width)
{
    // Using shared memory for tiles
    __shared__ int tileA[32][32];
    __shared__ int tileB[32][32];

    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int sum = 0;

    for (int m = 0; m < width / 32; ++m) {
        // Load elements into shared memory
        if (row < width && (m * 32 + threadIdx.x) < width)
            tileA[threadIdx.y][threadIdx.x] = a[row * width + m * 32 + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0;
        
        if (col < width && (m * 32 + threadIdx.y) < width)
            tileB[threadIdx.y][threadIdx.x] = b[(m * 32 + threadIdx.y) * width + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0;

        // Synchronize threads to ensure all elements are loaded
        __syncthreads();

        // Compute partial results for this tile
        for (int k = 0; k < 32; ++k)
            sum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];

        // Synchronize threads to prepare for loading the next tile
        __syncthreads();
    }

    // Write the result
    if (row < width && col < width)
        c[row * width + col] = sum;
}