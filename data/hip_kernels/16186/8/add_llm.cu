#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    // Use shared memory for faster data access
    __shared__ int shared_a[256];
    __shared__ int shared_b[256];

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int local_index = threadIdx.x;

    // Load data into shared memory
    shared_a[local_index] = a[index];
    shared_b[local_index] = b[index];
    __syncthreads();

    // Perform addition using shared memory
    c[index] = shared_a[local_index] + shared_b[local_index];
}