#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _cuda_add_scalar(int *in, int scalar, int n)
{
    // Calculate the global index of the thread
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Unrolling the loop for better performance
#pragma unroll
    for (int idx = globalIdx; idx < n; idx += blockDim.x * gridDim.x)
    {
        in[idx] += scalar; // Update the array element by adding the scalar
    }
}