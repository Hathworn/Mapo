#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Use shared memory and check thread within block bounds for better memory access patterns
    extern __shared__ float sharedImage[];

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < h && j < w) { // Ensure thread within image dimensions
        int pos = i * w + j;
        sharedImage[threadIdx.y * blockDim.x + threadIdx.x] = value; // Write to shared memory
        __syncthreads(); // Synchronize threads within block
        image[pos] = sharedImage[threadIdx.y * blockDim.x + threadIdx.x]; // Copy from shared to global memory
    }
}