#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Use grid-stride loop for better scalability
    for (int pos = blockIdx.x * blockDim.x + threadIdx.x; pos < len; pos += blockDim.x * gridDim.x) {
        d_res[pos] = d_src[pos] * scale;
    }
}