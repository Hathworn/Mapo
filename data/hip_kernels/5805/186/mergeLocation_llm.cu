#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Use a stride loop for better memory coalescing and efficient utilization of GPU threads
    for (int ptidx = blockIdx.x * blockDim.x + threadIdx.x; ptidx < npoints; ptidx += blockDim.x * gridDim.x)
    {
        short2 loc = loc_[ptidx];
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}