#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel(int32_t n, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    float cur_weight = *cur_tot_weight;
    float val = 1.0f;

    // Prefetch common value outside the loop for efficiency
    float new_scale = (max_count > 0) ? max(cur_weight, (float)max_count) / max_count : 0.0f;
    val += new_scale - 1.0f;

    // Calculate global thread index
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop for better performance
    for (int32_t i = idx; i < n; i += blockDim.x * gridDim.x) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        quadratic[diag_idx] += val;
    }

    // Reduce condition checks for thread synchronization
    if (threadIdx.x == 0) {
        atomicAdd(&linear[0], val * prior_offset);
    }
}