#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define INF 1000000

using namespace std;

__global__ void RoyFloyd(int* matrix, int k, int N)
{
    int i = blockDim.y * blockIdx.y + threadIdx.y;
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N && j < N) { // Ensure indices are within bounds
        int ik = matrix[i * N + k];
        int kj = matrix[k * N + j];
        int ij = matrix[i * N + j];

        if (ik != INF && kj != INF && ik + kj < ij) {
            matrix[i * N + j] = ik + kj; // Update the matrix element if a shorter path is found
        }
    }
}