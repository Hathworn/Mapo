#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void daxpy_kernel(int n, double a, double *x, double *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use loop to allow coalesced access and increase occupancy
    for (int idx = i; idx < n; idx += gridDim.x * blockDim.x)
    {
        y[idx] = a * x[idx] + y[idx];
    }
}