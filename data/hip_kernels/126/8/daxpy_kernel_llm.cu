#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void daxpy_kernel(int n, double a, double * x, double * y)
{
    // Use shared memory for improved memory access pattern
    extern __shared__ double shared_x[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load x into shared memory
    if (i < n) {
        shared_x[threadIdx.x] = x[i];
        __syncthreads();
        
        // Perform computation
        y[i] = a * shared_x[threadIdx.x] + y[i];
    }
}