#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copySharedMem(float *odata, const float *idata)
{
    __shared__ float tile[TILE_DIM][TILE_DIM]; // Change to 2D array for better indexing

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[threadIdx.y + j][threadIdx.x] = idata[(y + j) * width + x]; // Use 2D indexing

    __syncthreads();

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[(y + j) * width + x] = tile[threadIdx.y + j][threadIdx.x]; // Use 2D indexing
}