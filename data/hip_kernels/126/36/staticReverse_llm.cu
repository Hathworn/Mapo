#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void staticReverse(int *d, int n)
{
    extern __shared__ int s[];
    int t = threadIdx.x;
    int tr = n - t - 1;

    // Load data into shared memory
    if (t < n) s[t] = d[t];
    __syncthreads();

    // Read from shared memory and write to global memory
    if (tr < n) d[t] = s[tr];
}