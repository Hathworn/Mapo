#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNoBankConflicts(float *odata, const float *idata)
{
    __shared__ float tile[TILE_DIM][TILE_DIM+1];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    // Load input data into shared memory efficiently
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[threadIdx.y + j][threadIdx.x] = idata[(y + j) * width + x];

    __syncthreads();

    // Calculate new transposed positions
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    // Store transposed data from shared memory to output
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[(y + j) * width + x] = tile[threadIdx.x][threadIdx.y + j];
}