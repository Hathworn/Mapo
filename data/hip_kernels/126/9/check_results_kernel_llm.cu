#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void check_results_kernel(int n, double correctvalue, double *x)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use conditional assignment to reduce the number of branches
    bool incorrect = (i < n) && (x[i] != correctvalue);

    if (incorrect)
    {
        printf("ERROR at index = %d, expected = %f, actual: %f\n", i, correctvalue, x[i]);
    }
}