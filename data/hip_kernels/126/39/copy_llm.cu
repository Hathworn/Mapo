#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy(float *odata, const float *idata)
{
    // Calculate the global index
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    // Use shared memory for performance boost
    __shared__ float tile[TILE_DIM][TILE_DIM];
    
    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    {
        // Load data into shared memory
        if (y + j < gridDim.y * TILE_DIM && x < gridDim.x * TILE_DIM)
        {
            tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];
        }
        __syncthreads();
        
        // Store data from shared memory to global memory
        if (y + j < gridDim.y * TILE_DIM && x < gridDim.x * TILE_DIM)
        {
            odata[(y+j)*width + x] = tile[threadIdx.y+j][threadIdx.x];
        }
        __syncthreads();
    }
}