#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel launch configuration with fewer wasted threads
__global__ void init_data_kernel(int n, double* x)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate total threads in the grid
    for (; i < n; i += stride) // Loop for processing elements beyond initial set
    {
        x[i] = n - i;
    }
}