#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void spinKernel(unsigned long long timeout_clocks = 100000ULL)
{
    // Cache clock64 value to optimize loop
    register unsigned long long start_time = clock64();
  
    // Use a while condition to reduce branch instructions
    while(timeout_clocks == ~0ULL || clock64() - start_time <= timeout_clocks) {
        // Busy wait
    }
}