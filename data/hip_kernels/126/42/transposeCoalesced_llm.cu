#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeCoalesced(float *odata, const float *idata)
{
    __shared__ float tile[TILE_DIM][TILE_DIM + 1]; // Avoid bank conflicts

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    // Coalesced read from global memory
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
        tile[threadIdx.y + j][threadIdx.x] = idata[(y + j) * width + x];
    }

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x; // Transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    // Coalesced write to global memory
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
        odata[(y + j) * width + x] = tile[threadIdx.x][threadIdx.y + j];
    }
}