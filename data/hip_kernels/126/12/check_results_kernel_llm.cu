#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void check_results_kernel(int n, double correctvalue, double* x) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure the thread index is within array bounds
    if (i < n) {
        // Compare using epsilon to handle floating-point comparison issues
        if (fabs(x[i] - correctvalue) > 1e-9) {
            printf("ERROR at index = %d, expected = %f, actual: %f\n", i, correctvalue, x[i]);
        }
    }
}