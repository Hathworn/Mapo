#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNaive(float *odata, const float *idata)
{
    // Calculate x and y indices
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    // Use shared memory to optimize global memory access
    __shared__ float tile[TILE_DIM][TILE_DIM+1];

    // Load data into shared memory
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        tile[threadIdx.y + j][threadIdx.x] = idata[(y + j) * width + x];
    }

    __syncthreads();

    // Transpose the tile
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        odata[(x * width) + (y + j)] = tile[threadIdx.x][threadIdx.y + j];
    }
}