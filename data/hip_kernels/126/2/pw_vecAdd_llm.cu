#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_vecAdd(float *y, float *a, float *b, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll loop by processing multiple elements per thread to enhance performance
    int stride = gridDim.x * blockDim.x;
    for (int idx = i; idx < n; idx += stride) {
        y[idx] = a[idx] + b[idx];
    }
}