#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_tanh(float *y, float *a, int n) {
    // Use grid stride loop to handle workloads larger than the block size
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        y[i] = tanh(a[i]);
    }
}