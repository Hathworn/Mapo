#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_vecMul(float *y, float *a, float *b, int n) {
    // Use stride loop for handling large arrays efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    while (i < n) {
        y[i] = a[i] * b[i];
        i += stride;
    }
}