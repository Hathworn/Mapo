#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dynamicReverse(int *d, int n)
{
    extern __shared__ int s[];
    int t = threadIdx.x;
    int tr = n - t - 1;

    // Load data into shared memory, ensuring bounds are respected
    if (t < n) {
        s[t] = d[t];
    }
    __syncthreads();

    // Reverse the array and write it back, ensuring bounds are respected
    if (t < n) {
        d[t] = s[tr];
    }
}