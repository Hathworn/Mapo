#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_data_kernel(int n, double* x)
{
    // Calculate the global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use strided loop for better data access pattern
    for (; i < n; i += blockDim.x * gridDim.x)
    {
        x[i] = n - i;
    }
}