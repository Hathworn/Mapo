#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_biasAdd(float *y, float *bias, int n, int nBias) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop over elements in a grid-strided manner
    for (; i < n; i += stride) {
        y[i] += bias[i % nBias];
    }
}