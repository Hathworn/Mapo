#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *a, int offset)
{
    int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
    float x = (float)i;

    // Precompute sin and cos to avoid redundant calculations
    float s = sinf(x);
    float c = cosf(x);

    // Use hypotf for computing sqrt(s*s + c*c) directly (optimizes numerical stability)
    a[i] = a[i] + hypotf(s, c);
}