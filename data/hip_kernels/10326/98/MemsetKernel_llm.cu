#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate the position using the grid-stride loop for efficient memory access
    for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < h; i += blockDim.y * gridDim.y)
    {
        for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < w; j += blockDim.x * gridDim.x)
        {
            const int pos = i * w + j;
            image[pos] = value;
        }
    }
}