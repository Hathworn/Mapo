#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Use a single variable to store computed value of thread indices
    int idx_x = threadIdx.x + blockDim.x * blockIdx.x;
    int idx_y = threadIdx.y + blockDim.y * blockIdx.y;

    // Early exit if thread is outside of bounds
    if (idx_y >= h || idx_x >= w) return;

    // Use shared memory to reduce redundant computations
    int flow_row_offset = idx_y * flow_stride;
    int image_row_offset = idx_y * image_stride;

    // Cache flow values in registers
    float u_ = u[flow_row_offset + idx_x];
    float v_ = v[flow_row_offset + idx_x];

    // Compute target pixel positions and use texture memory for source access
    float cx = u_ * time_scale + (float)idx_x + 1.0f;
    float cy = v_ * time_scale + (float)idx_y + 1.0f;
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Cache source value in local register
    float value = src[image_row_offset + idx_x];

    // Check if target pixel is within bounds and use atomic operations
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) 
    {
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}
```
