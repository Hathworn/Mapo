#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate coordinates and integer parts efficiently
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py, dx = modff(cx, &px), dy = modff(cy, &py);
    int tx = (int)px, ty = (int)py;

    float value = src[image_row_offset + j];
    float weight;
    
    // Combine bounds check in a single branch
    if (tx < w && tx >= 0 && ty < h && ty >= 0)
    {
        weight = dx * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Pre-calculate common sub-expressions
    float dx_1 = 1.0f - dx;
    float dy_1 = 1.0f - dy;

    tx -= 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0)
    {
        weight = dx_1 * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    ty -= 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0)
    {
        weight = dx_1 * dy_1;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    tx += 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0)
    {
        weight = dx * dy_1;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}