#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Use single calculation of thread index
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid unnecessary computation when out of bounds
    if (i < h && j < w) 
    {
        const int pos = i * s + j;
        float scale = normalization_factor[pos];
        float invScale = __fdividef(1.0f, scale); // Use fast divide function
        image[pos] *= (scale == 0.0f ? 1.0f : invScale);
    }
}