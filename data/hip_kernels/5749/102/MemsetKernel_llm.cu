#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate the unique thread index
    int idx = blockDim.x * blockDim.y * blockIdx.y * gridDim.x + blockIdx.x * blockDim.x * blockDim.y 
              + threadIdx.y * blockDim.x + threadIdx.x;
              
    // Check for out-of-bounds condition
    if (idx >= w * h) return;

    // Set the value at the calculated linear index
    image[idx] = value;
}