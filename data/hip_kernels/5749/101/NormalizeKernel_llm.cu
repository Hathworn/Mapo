#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Ensure thread processes only valid elements
    if (i < h && j < w) 
    {
        const int pos = i * s + j;  // Compute position once

        // Directly compute inverse scale
        float invScale = (normalization_factor[pos] == 0.0f) ? 1.0f : (1.0f / normalization_factor[pos]);

        // Apply normalization
        image[pos] *= invScale;
    }
}