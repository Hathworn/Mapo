#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i < h && j < w)  // Combine boundary checks into one condition
    {
        int flow_row_offset = i * flow_stride;
        int image_row_offset = i * image_stride;

        float u_ = u[flow_row_offset + j];
        float v_ = v[flow_row_offset + j];

        // Calculate bottom left corner of target pixel once
        float cx = __fmul_rn(u_, time_scale) + __fadd_rn((float)j, 1.0f);
        float cy = __fmul_rn(v_, time_scale) + __fadd_rn((float)i, 1.0f);
        // Convert corner position to integer representation of pixel
        int tx = __float2int_rn(cx);
        int ty = __float2int_rn(cy);

        float value = src[image_row_offset + j];
        // Only perform atomicAdd if tx and ty are within valid bounds
        if (tx >= 0 && tx < w && ty >= 0 && ty < h)
        {
            atomicAdd(dst + ty * image_stride + tx, value);
        }
    }
}