#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void createLookupKernel(const int* inds, int total, int* output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread index is within bounds
    if (idx >= total) return;
    
    // Use atomic operation to prevent write conflicts
    atomicExch(&output[inds[idx]], idx);
}