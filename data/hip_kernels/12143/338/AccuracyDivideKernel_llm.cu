#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use blockIdx.x and threadIdx.x to access the first element only if they are (0,0).
    // This avoids unnecessary global memory accesses and ensures only one thread modifies *accuracy.
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        *accuracy /= N;
    }
}