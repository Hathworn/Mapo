#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

__global__ void vsub(const float *a, const float *b, float *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) // Ensure thread operates within bounds
    {
        c[i] = a[i] - b[i];
    }
}

}