#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

}

__global__ void vmul(const float *a, const float *b, float *c, int n) 
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within the bounds of the arrays
    if (i < n) {
        c[i] = a[i] * b[i];
    }
}