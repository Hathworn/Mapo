#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

}

__global__ void vdiv(const float *a, const float *b, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if the index is within bounds
    if (i < gridDim.x * blockDim.x) {
        c[i] = a[i] / b[i];
    }
}