#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16 // Define BLOCK_SIZE for matrix blocks

__global__ void matMultCUDA(const float* a, int lda, const float* b, int ldb, float* c, int ldc, int n)
{
    __shared__ float matA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float matB[BLOCK_SIZE][BLOCK_SIZE];
    const int tidc = threadIdx.x;
    const int tidr = threadIdx.y;
    const int bidc = blockIdx.x*BLOCK_SIZE;
    const int bidr = blockIdx.y*BLOCK_SIZE;
    
    float results = 0;
    float comp = 0;
    
    for (int j = 0; j < n; j += BLOCK_SIZE) {
        // Load data into shared memory with coalesced accesses
        matA[tidr][tidc] = a[(tidr + bidr) * lda + tidc + j];
        matB[tidr][tidc] = b[(tidr + j) * ldb + tidc + bidc];

        __syncthreads();

        // Compute in register and reduce warp-level synchronization
        #pragma unroll
        for (int i = 0; i < BLOCK_SIZE; i++) {
            float t, product;
            product = matA[tidr][i] * matB[i][tidc];
            comp -= product;
            t = results - comp;
            comp = (t - results) + comp;
            results = t;
        }

        __syncthreads();
    }
    
    // Write results back to global memory
    c[(tidr + bidr) * ldc + tidc + bidc] = results;
}