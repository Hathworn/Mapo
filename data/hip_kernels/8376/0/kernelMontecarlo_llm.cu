#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define N 128*128

__global__ void kernelMontecarlo(float *x, float *y, int *contador) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D indexing for simplicity
    if (idx < N) { // Ensure index is within bounds
        float x_val = x[idx];
        float y_val = y[idx];
        if ((x_val * x_val + y_val * y_val) <= 1.0f) { // Check condition
            atomicAdd(contador, 1); // Safely increment counter
        }
    }
}