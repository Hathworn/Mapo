#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelMultMatrices(float *a, float *b, float *c, int m, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < n && j < m) { // Ensure threads access valid indices
        float sum = 0.0f; // Use a register for accumulation
        for (int k = 0; k < N; k++) {
            sum += a[j + k * n] * b[k + i * n];
        }
        c[j + i * n] = sum; // Store result in the output matrix
    }
    // Removed __syncthreads() as it's unnecessary here
}