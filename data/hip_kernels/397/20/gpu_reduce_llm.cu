#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_reduce(int *c, int size)
{
    // Get unique global thread ID
    int position = threadIdx.x + blockIdx.x * blockDim.x;

    // Check bounds and reduce in place, handle odd size with additional check
    if (position < size / 2) 
    {
        int partner_idx = position + size / 2;
        if (c[position] < c[partner_idx]) 
        {
            c[position] = c[partner_idx];
        }
    }
    else if (size % 2 != 0 && position == size - 1) 
    {
        if (c[position] > c[size / 2 - 1]) 
        {
            c[size / 2 - 1] = c[position];
        }
    }
}