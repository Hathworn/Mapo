#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

typedef struct {
    int y;
    int x;
} Antena;

#define m(y,x) mapa[ (y * cols) + x ]

#define TAMBLOCK 128

__global__ void gpu_reduce(int *c, int size)
{
    // Calculate global thread ID
    int position = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds
    if (position < size / 2) {
        // Reduce using threads efficiently
        if (c[position] < c[position + size / 2]) {
            c[position] = c[position + size / 2];
        }
    }

    // Handle case where size is odd separately
    if (position == 0 && size % 2 != 0) {
        if (c[0] < c[size - 1]) {
            c[0] = c[size - 1];
        }
    }
}