#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

/**
* Estructura antena
*/
typedef struct {
    int y;
    int x;
} Antena;

/**
* Macro para acceder a las posiciones del mapa
*/
#define m(y,x) mapa[ (y * cols) + x ]

/**
* Definimos el tamaño de bloque
*/
#define TAMBLOCK 128

/**
* Inicializa el mapa con el valor máximo
*/
__global__ void gpu_init(int *mapad, int max, int size) {
    int position = blockIdx.x * blockDim.x + threadIdx.x + 
                   blockIdx.y * blockDim.y * gridDim.x + threadIdx.y * gridDim.x * blockDim.x;

    // Only initialize positions within the map size
    if (position < size) {
        mapad[position] = max;
    }
}