#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_init(int *mapad, int max, int size)
{
    // Calculate position using block and thread indices
    int position = blockDim.x * blockDim.y * blockIdx.y * gridDim.x 
                 + blockDim.x * blockDim.y * blockIdx.x 
                 + blockDim.x * threadIdx.y 
                 + threadIdx.x;

    // Ensure position is within bounds
    if (position < size) {
        mapad[position] = max;
    }
}