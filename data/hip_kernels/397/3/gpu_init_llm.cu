#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_init(int *mapad, int max, int size)
{
    // Calculate global position using block and thread indices
    int position = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    // Check if position is within array bounds and initialize
    if (position < size) {
        mapad[position] = max;
    }
}