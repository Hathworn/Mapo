#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_init(int *mapad, int max, int size)
{
    // Calculate global thread position in flattened array
    int threadId = blockIdx.y * gridDim.x * blockDim.x * blockDim.y + blockIdx.x * blockDim.x * blockDim.y +
                   threadIdx.y * blockDim.x + threadIdx.x;

    // Assign value to array position if within bounds
    if (threadId < size) {
        mapad[threadId] = max;
    }
}