#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_reduce(int *c, int size)
{
    // Calculate global thread position
    int position = blockDim.x * blockDim.y * ((blockIdx.y * gridDim.x) + blockIdx.x) + ((threadIdx.y * blockDim.x) + threadIdx.x);

    // Check if position is within bounds
    if (position < size / 2) {  // Only process half the size
        // Perform reduction for even-sized arrays
        if (c[position] < c[position + size / 2]) {
            c[position] = c[position + size / 2];
        }
    }

    // Handle odd size by checking last element separately
    if (size % 2 != 0 && position == 0) {  // Only the first thread checks this
        if (c[0] < c[size - 1]) {
            c[0] = c[size - 1];
        }
    }
}