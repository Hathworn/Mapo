#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_init(int *mapad, int max, int size)
{
    // Calculate global linear thread index
    int position = blockIdx.x * blockDim.x * blockDim.y 
                 + blockIdx.y * gridDim.x * blockDim.x * blockDim.y 
                 + threadIdx.y * blockDim.x + threadIdx.x;

    // Bounds check
    if (position < size) {
        mapad[position] = max;
    }
}