#include "hip/hip_runtime.h"
```cuda
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_init(int *mapad, int max, int size){

    // Calculate the global thread index
    int position = blockIdx.x * blockDim.x + threadIdx.x + 
                   blockIdx.y * blockDim.x * gridDim.x;

    // Initialize mapad if position is within bounds
    if (position < size) {
        mapad[position] = max;
    }
}