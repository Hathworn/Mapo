#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_init(int *mapad, int max, int size){
    // Compute global thread index
    int position = blockIdx.y * blockDim.y * gridDim.x * blockDim.x 
                 + blockIdx.x * blockDim.x
                 + threadIdx.y * blockDim.x 
                 + threadIdx.x;

    // Initialize if within bounds
    if(position < size) {
        mapad[position] = max;
    }
}