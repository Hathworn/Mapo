#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_init(int *mapad, int max, int size) {

    // Calculate global thread index
    int position = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y +
                   threadIdx.y * blockDim.x + threadIdx.x;

    // Initialize if position is within bounds
    if (position < size) {
        mapad[position] = max;
    }
}