#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_reduce(int *c, int size)
{
    // Calculate the global position of the thread
    int position = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * (blockIdx.x + gridDim.x * blockIdx.y));

    // Optimized boundary check for in-bounds threads
    if (position < size / 2) {
        int offset = size / 2;

        // Perform reduction using the current thread's pair
        if (c[position] < c[position + offset]) {
            c[position] = c[position + offset];
        }
    }

    // Handle odd size case with a single additional comparison
    if (size % 2 != 0 && position == 0) {
        if (c[0] < c[size - 1]) {
            c[0] = c[size - 1];
        }
    }
}