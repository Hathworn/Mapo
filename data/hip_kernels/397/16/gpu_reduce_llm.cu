#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_reduce(int *c, int size)
{
    // Calculate global thread index
    int position = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform reduction if within bounds
    if (position < size/2) {
        int offset = size % 2;
        
        // If array size is odd, compare with last element
        if (offset && (position == size/2 - 1)) {
            if (c[position] < c[size-1]) {
                c[position] = c[size-1];
            }
        } else {
            // Compare with corresponding element at position + size/2
            if (c[position] < c[position + size/2]) {
                c[position] = c[position + size/2];
            }
        }
    }
}