#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_reduce(int *c, int size)
{
    // Calculate unique thread index across the grid
    int idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    
    // Each thread reduces a pair of elements, if within size
    if (idx < size / 2)
    {
        if (c[idx] < c[idx + size / 2])
        {
            c[idx] = c[idx + size / 2];
        }
    }

    // Handle case for odd size by comparing the last element
    if (size % 2 != 0 && idx == 0)
    {
        if (c[0] < c[size - 1])
        {
            c[0] = c[size - 1];
        }
    }
}