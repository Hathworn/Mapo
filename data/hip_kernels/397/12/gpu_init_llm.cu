#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_init(int *mapad, int max, int size)
{
    // Calculate global thread position more efficiently
    int position = threadIdx.x + blockIdx.x * blockDim.x 
                 + (threadIdx.y + blockIdx.y * blockDim.y) * gridDim.x * blockDim.x; 

    if (position < size) 
    {
        mapad[position] = max;
    }
}