#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAT_TYPE double
#define MAT_SIZE 1024
#define N MAT_SIZE
#define N2 MAT_SIZE*MAT_SIZE

#define BLOCK 256
#define THREAD 512

void stopwatch(int);

__global__ void cuda_mul(MAT_TYPE* A, MAT_TYPE* B, MAT_TYPE* C, int w)
{
    // Calculate thread row and column within the matrix
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = ty * w + tx;

    if (tx < w && ty < w) // Ensure within bounds
    {
        MAT_TYPE v = 0;

        // Use shared memory for efficient data reuse
        __shared__ MAT_TYPE As[THREAD];
        __shared__ MAT_TYPE Bs[THREAD];

        for (int i = 0; i < w; i += BLOCK)
        {
            As[threadIdx.y * blockDim.x + threadIdx.x] = A[ty * w + (i + threadIdx.x)];
            Bs[threadIdx.y * blockDim.x + threadIdx.x] = B[(i + threadIdx.y) * w + tx];

            __syncthreads(); // Synchronize to ensure all data is loaded into shared memory

            for (int j = 0; j < BLOCK && (i + j) < w; ++j)
            {
                v += As[threadIdx.y * blockDim.x + j] * Bs[j * blockDim.x + threadIdx.x];
            }

            __syncthreads(); // Synchronize before loading new data into shared memory
        }

        C[tid] = v;
    }
}