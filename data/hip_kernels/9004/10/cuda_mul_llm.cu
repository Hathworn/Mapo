#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_mul(int* A, int* B, int* C, int w)
{
    // Calculate global thread index
    int tx = blockDim.x * blockIdx.x + threadIdx.x;
    int ty = blockDim.y * blockIdx.y + threadIdx.y;

    if (tx < w && ty < w) { // Ensure tx and ty are within bounds
        int tid = w * ty + tx;
        int v = 0;

        for (int i = 0; i < w; i++)
        {
            int a = A[ty * w + i];
            int b = B[i * w + tx];
            v += a + b; // Calculate sum instead of multiplication
        }

        C[tid] = v;
    }
}