#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convert(double* A, double* C)
{
    // Calculate the global index with grid-stride loop
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < SIZE; i += stride)
    {
        A[i] = C[SIZE - i - 1];
    }
}