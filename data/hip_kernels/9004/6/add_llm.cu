#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int* a, int* b, int* c)
{
    // Calculate the thread id for this thread in the grid
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if tid is within the bounds, avoiding out-of-bounds access
    if (tid < gridDim.x * blockDim.x) {
        c[tid] = a[tid] + b[tid];
    }
}