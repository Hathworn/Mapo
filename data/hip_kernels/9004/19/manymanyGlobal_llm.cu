#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void manymanyGlobal(int* a, int* b)
{
    // Get global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Iterate over the iterations
    for(int j = 0; j < ITER; j++)
    {
        // Use stride loop based on grid size for better parallelism
        for(int i = idx; i < SIZE; i += blockDim.x * gridDim.x)
        {
            a[i] = 0;
            b[i] = 0;
        }
    }
}