#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global index for 1D array
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (y < h && x < w) {
        int pos = y * w + x;
        image[pos] = value; // Assign the value to the calculated position
    }
}