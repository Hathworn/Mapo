#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // Unroll modff computation
    float px = floorf(cx), py = floorf(cy);
    float dx = cx - px, dy = cy - py;

    // Use single load for value
    float value = src[image_row_offset + j];
    float weight;
    
    const int coordinates[4][2] = {
        {0, 0}, // bottom right
        {-1, 0}, // bottom left
        {-1, -1}, // upper left
        {0, -1}  // upper right
    };
    
    const float weights[4] = {
        dx * dy,
        (1.0f - dx) * dy,
        (1.0f - dx) * (1.0f - dy),
        dx * (1.0f - dy)
    };

    // Combine operations into a single block
    for (int idx = 0; idx < 4; ++idx)
    {
        int tx = (int)px + coordinates[idx][0];
        int ty = (int)py + coordinates[idx][1];
        if (!(tx >= w || tx < 0 || ty >= h || ty < 0))
        {
            weight = weights[idx];
            _atomicAdd(dst + ty * image_stride + tx, value * weight);
            _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
        }
    }
}