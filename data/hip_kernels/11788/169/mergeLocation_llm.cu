#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate global thread index based on grid and block dimensions
    int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride variable for loop, allowing more threads beyond grid size to process if needed
    int stride = gridDim.x * blockDim.x; 

    // Loop through points with stride to cover entire data array if needed
    for (; ptidx < npoints; ptidx += stride) {
        short2 loc = loc_[ptidx];
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}