#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < h && j < w) { // Combined boundary check into a single condition
        const int pos = i * s + j;
        float scale = normalization_factor[pos];
        float invScale = (scale != 0.0f) ? (1.0f / scale) : 1.0f; // Switched order for improved readability
        image[pos] *= invScale;
    }
}