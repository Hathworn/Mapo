#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Early return to avoid out-of-bound access
    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Calculate target pixel coordinates
    float cx = u_ * time_scale + (float)j + 1.0f;
    float cy = v_ * time_scale + (float)i + 1.0f;

    // Avoid multiple type casts and temporary variables
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    float value = src[image_row_offset + j];

    // Ensure tx and ty are within bounds
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}