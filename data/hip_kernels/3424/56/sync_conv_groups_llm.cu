#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Synchronize all threads in the grid using a dummy kernel
__global__ void sync_conv_groups() {
    // Improved use of built-in sync function
    __syncthreads(); // Synchronize threads within a block
}