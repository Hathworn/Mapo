#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel function
__global__ void var(int *a, int *b, int n, float mean) {
    // Get global thread index
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    float sum = 0;

    // Parallelize operations across threads
    for (int i = globalIdx; i < n; i += stride) {
        float diff = a[i] - mean;
        sum += diff * diff;
    }

    // Use shared memory to reduce sums within block
    extern __shared__ float sharedSum[];

    sharedSum[threadIdx.x] = sum;
    __syncthreads();

    // Perform parallel reduction within the block
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sharedSum[threadIdx.x] += sharedSum[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        b[blockIdx.x] = sharedSum[0];
    }
}