#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void maxi(int *a, int *b, int n)
{
    // Calculate the index of the current thread within the block
    int tid = threadIdx.x;
    // Calculate the global index of the current thread
    int index = 256 * blockIdx.x + tid;
    // Shared memory for block-wide maximum computation
    __shared__ int shared_max[256];
    
    // Initialize shared memory
    shared_max[tid] = (index < n) ? a[index] : INT_MIN;
    __syncthreads();
    
    // Perform reduction to find maximum within the block
    for (int stride = 128; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_max[tid] = max(shared_max[tid], shared_max[tid + stride]);
        }
        __syncthreads();
    }
    
    // Write the maximum value found by this block to the result array
    if (tid == 0) {
        b[blockIdx.x] = shared_max[0];
    }
}