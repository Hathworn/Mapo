#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum(int *a, int *b, int n)
{
    __shared__ int shared_sum[256]; // Allocate shared memory
    int tid = threadIdx.x;
    int block_offset = blockDim.x * blockIdx.x;
    shared_sum[tid] = 0;

    // Load elements into shared memory
    for (int i = tid + block_offset; i < min(block_offset + blockDim.x, n); i += blockDim.x)
    {
        shared_sum[tid] += a[i];
    }

    __syncthreads();  // Synchronize threads before reduction

    // Reduce within the block
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            shared_sum[tid] += shared_sum[tid + stride];
        }
        __syncthreads();  // Ensure all threads reach this point before next iteration
    }

    // Write the result of block reduction to global memory
    if (tid == 0)
    {
        b[blockIdx.x] = shared_sum[0];
    }
}