#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mini1(int *a,int *b,int n)
{
    // Use shared memory for partial results
    __shared__ int shared_min[256];

    int idx = threadIdx.x;
    int globalIdx = blockIdx.x * 256 + idx;

    // Initialize shared memory
    shared_min[idx] = INT_MAX;

    // Load data into shared memory if within bounds
    if (globalIdx < n) {
        shared_min[idx] = a[globalIdx];
    }
    __syncthreads();

    // Perform parallel reduction in shared memory
    for (int stride = 128; stride > 0; stride /= 2) {
        if (idx < stride && (idx + stride) < 256) {
            shared_min[idx] = min(shared_min[idx], shared_min[idx + stride]);
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (idx == 0) {
        b[blockIdx.x] = shared_min[0];
    }
}