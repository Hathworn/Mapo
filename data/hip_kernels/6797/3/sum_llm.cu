#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void sum(int *a, int *b, int n)
{
    // Calculate unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;

    // Loop stride to handle array elements
    for (int i = idx; i < n; i += gridDim.x * blockDim.x)
    {
        sum += a[i];
    }

    // Atomic add to avoid race conditions when writing to b
    atomicAdd(&b[blockIdx.x], sum);
}