#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use block and thread indices to ensure parallel access to accuracy
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        accuracy[idx] /= N;  // Ensure each element is processed individually
    }
}