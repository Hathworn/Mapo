#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_grid_array_to_value(float *arr, float value, int N_grid) {
    // Calculate 1D index considering 3D grid and block dimensions
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;

    // Linearized index calculation
    int index = idz * N_grid * N_grid + idy * N_grid + idx;

    // Boundary check and setting value
    if (idx < N_grid && idy < N_grid && idz < N_grid) {
        arr[index] = value;
    }
}