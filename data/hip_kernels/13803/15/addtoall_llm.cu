#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addtoall(int* a, int b)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index within bounds (assuming the calling code uses appropriate block/grid dimensions)
    if (i < N) { 
        atomicAdd(&(a[i]), b);
    }
}