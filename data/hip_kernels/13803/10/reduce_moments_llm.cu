#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_moments(float *d_arr, float *d_results, int N)
{
    __shared__ float sh_array[pThreads];
    int n = blockDim.x * blockIdx.x + threadIdx.x;

    // Load elements into shared memory
    if (n < N) {
        sh_array[threadIdx.x] = d_arr[n]; // Corrected - load from global to shared memory
    } else {
        sh_array[threadIdx.x] = 0; // Initialize with 0 if outside array bounds
    }
    __syncthreads();

    // Perform reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sh_array[threadIdx.x] += sh_array[threadIdx.x + s]; // Accumulate within shared memory
        }
        __syncthreads();
    }

    // Write result of this block to global memory
    if (threadIdx.x == 0) {
        d_results[blockIdx.x] = sh_array[0];
    }
}