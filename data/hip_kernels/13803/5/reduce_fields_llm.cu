#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce_fields(float *d_rho, float *d_Ex, float* d_Ey, float* d_Ez, float *d_Rrho, float* d_REx, float* d_REy, float* d_REz, int N)
{
    __shared__ float rho_array[gThreadsAll];
    __shared__ float Ex_array[gThreadsAll];
    __shared__ float Ey_array[gThreadsAll];
    __shared__ float Ez_array[gThreadsAll];

    int n = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Load data to shared memory only if the index is within range
    if (n < N) {
        rho_array[threadIdx.x] = d_rho[n];
        Ex_array[threadIdx.x] = d_Ex[n] * d_Ex[n];
        Ey_array[threadIdx.x] = d_Ey[n] * d_Ey[n];
        Ez_array[threadIdx.x] = d_Ez[n] * d_Ez[n];
    } else {
        rho_array[threadIdx.x] = 0.0f;
        Ex_array[threadIdx.x] = 0.0f;
        Ey_array[threadIdx.x] = 0.0f;
        Ez_array[threadIdx.x] = 0.0f;
    }
    __syncthreads();

    // Perform reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            rho_array[threadIdx.x] += rho_array[threadIdx.x + s];
            Ex_array[threadIdx.x] += Ex_array[threadIdx.x + s];
            Ey_array[threadIdx.x] += Ey_array[threadIdx.x + s];
            Ez_array[threadIdx.x] += Ez_array[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Write result of reduction for this block to global memory
    if (threadIdx.x == 0) {
        d_Rrho[blockIdx.x] = rho_array[0];
        d_REx[blockIdx.x] = Ex_array[0];
        d_REy[blockIdx.x] = Ey_array[0];
        d_REz[blockIdx.x] = Ez_array[0];
    }
}