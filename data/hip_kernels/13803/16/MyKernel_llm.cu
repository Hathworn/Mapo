#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MyKernel(float* devPtr, size_t pitch, int width, int height)
{
    // Calculate row using blockIdx.y and threadIdx.y
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    if (r < height) 
    {
        float* row = (float*)((char*)devPtr + r * pitch);
        // Calculate column using blockIdx.x and threadIdx.x
        for (int c = blockIdx.x * blockDim.x + threadIdx.x; c < width; c += blockDim.x * gridDim.x) 
        {
            row[c] = 17.3;
        }
    }
}