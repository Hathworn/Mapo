#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_down_after_fft(float *d_Ex, float *d_Ey, float *d_Ez, int N_grid, int N_grid_all) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;
    int index = k * N_grid * N_grid + j * N_grid + i;

    // Simplify condition check and scale down only when inside grid bounds.
    if (index < N_grid * N_grid * N_grid) {
        float scale_factor = 1.0f / N_grid_all; // Calculate scale factor once.
        d_Ex[index] *= scale_factor;
        d_Ey[index] *= scale_factor;
        d_Ez[index] *= scale_factor;
    }
}