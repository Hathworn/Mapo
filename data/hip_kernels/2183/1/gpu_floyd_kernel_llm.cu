#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_floyd_kernel(int k, int* adjacency_mtx, int* paths, int size)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col >= size) return;
    int idx = size * blockIdx.y + col;

    // Use shared memory to store the row elements to avoid repeated global memory access
    __shared__ int shared_row[1024]; // Assume maximum blockDim.x is 1024 or adjust accordingly

    if (threadIdx.x == 0) {
        shared_row[threadIdx.x] = adjacency_mtx[size * blockIdx.y + k];
    }
    __syncthreads();

    int best = shared_row[0];
    if (best == INF) return;

    int tmp_b = adjacency_mtx[k * size + col];
    if (tmp_b == INF) return;

    int cur = best + tmp_b;
    if (cur < adjacency_mtx[idx]) {
        adjacency_mtx[idx] = cur;
        paths[idx] = k;
    }
}