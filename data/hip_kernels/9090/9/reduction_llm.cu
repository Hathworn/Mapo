#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction(const int N, float *a, float *result) {
    int thread = threadIdx.x;
    int block  = blockIdx.x;
    int blockSize = blockDim.x;
    int gridSize = gridDim.x;

    //unique global thread ID
    int id = thread + block*blockSize;

    __shared__ float s_sum[256];

    // Use loop unrolling for better performance
    float sum = 0;
    for (int i=0; i<4; i++) {
        int index = id + i * blockSize * gridSize;
        if(index < N){
            sum += a[index];
        }
    }
    s_sum[thread] = sum;

    __syncthreads(); // Ensure all sums are stored in shared memory

    #pragma unroll // Optimization: Unroll the reduction loop
    for (int offset = blockSize / 2; offset > 0; offset /= 2) {
        if (thread < offset) {
            s_sum[thread] += s_sum[thread + offset];
        }
        __syncthreads(); // Ensure all threads complete the reduction step
    }

    // Write block result to the output array
    if (thread == 0) {
        result[block] = s_sum[0];
    }
}