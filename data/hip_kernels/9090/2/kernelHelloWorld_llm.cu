#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelHelloWorld() {
    // Combine block and thread index for global thread index
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

    printf("Hello World from global thread ID %d, thread %d of block %d!\n", globalThreadId, threadIdx.x, blockIdx.x);
}