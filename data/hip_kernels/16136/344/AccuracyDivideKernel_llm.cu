#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Optimize by checking if N is not zero to prevent division by zero
    if (N != 0) {
        *accuracy /= N;
    }
}