#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce(int *a, int *b, int n) {
    __shared__ int sdata[1024]; // Allocate shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load elements into shared memory
    sdata[tid] = (i < n) ? a[i] : 0;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result back to global memory
    if (tid == 0) {
        atomicAdd(b, sdata[0]);
    }
}