#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cpy(float *a, float *b, int n) {
    // Use shared memory for coalesced access
    extern __shared__ float shared_b[];
    
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tid = threadIdx.x;
    
    if (i < n) {
        // Load data into shared memory
        shared_b[tid] = b[i];
        __syncthreads(); // Synchronize to make sure all shared memory loads are complete

        // Copy from shared memory to global memory
        a[i] = shared_b[tid];
    }
}