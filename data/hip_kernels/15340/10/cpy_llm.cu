#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cpy(int *a, int *b, int n) {
    // Calculate global thread index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Declare shared memory for partial sums
    __shared__ int sharedSum[256];

    // Each thread calculates its sum
    int localSum = 0;
    while (i < n) {
        localSum += b[i];
        i += blockDim.x * gridDim.x;
    }

    // Store local sum in shared memory
    sharedSum[threadIdx.x] = localSum;

    // Synchronize threads in the block
    __syncthreads();

    // Reduce partial sums within this block
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sharedSum[threadIdx.x] += sharedSum[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Add block's result to global sum using atomicAdd
    if (threadIdx.x == 0) {
        atomicAdd(a, sharedSum[0]);
    }
}