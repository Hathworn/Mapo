#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Run_Me(int* The_Array, int size)
{
    // Use both blockIdx.x and threadIdx.x to parallelize the workload
    int ID = blockIdx.x * blockDim.x + threadIdx.x;
    if (ID < size) // Ensure ID is within bounds of the array
    {
        The_Array[ID] *= The_Array[ID]; // Optimize multiplication
    }
}