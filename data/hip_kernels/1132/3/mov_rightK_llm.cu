#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

void showMatriz(int *matriz, int anchura, int altura);
void generateSeeds(int *matriz, int ancho, int alto, int cantidad, char modo);
void gestionSemillas(int *matriz, int ancho, int numeroSemillas, int alto, char modo);
int checkFull(int *matriz, int tamano);
bool checkMove(int *matriz, int ancho, int alto);
void guardar(int vidas, int *tablero, int altura, int anchura, char dificultad);
int* cargar();
int* MostrarEspecificaciones();

hipError_t cudaStatus;

/*	stack_right
*	Shift non-zero elements to the right
*/
__device__ void stack_right(int *matriz, int anchura, int altura, int x, int y)
{
    for (int i = anchura - 1; i > 0; i--)
    {
        // Shift non-zero element if next is zero
        if ((y != anchura - 1) && (matriz[x*anchura + y] != 0) && matriz[x*anchura + (y + 1)] == 0)
        {
            matriz[x*anchura + (y + 1)] = matriz[x*anchura + y];
            matriz[x*anchura + y] = 0;
        }
        __syncthreads(); // Synchronize threads in the block
    }
}

/*	add_right
*	Add adjacent equal elements to the right
*/
__device__ void add_right(int *matriz, int x, int y, int altura, int anchura)
{
    if (y != anchura - 1 && y < anchura)
    {
        if (matriz[x*anchura + y] != 0)
        {
            if (matriz[x*anchura + y] == matriz[x*anchura + (y + 1)])
            {
                int iguales = 0;
                iguales++;
                for (int i = 1; y + i < anchura; i++)
                {
                    if (matriz[x*anchura + y] == matriz[x*anchura + (y + i)])
                    {
                        iguales++;
                    }
                    else {
                        break;
                    }
                }
                if (iguales % 2 == 0)
                {
                    matriz[x*anchura + (y + 1)] = matriz[x*anchura + (y + 1)] * 2;
                    matriz[x*anchura + y] = 0;
                }
            }
            else if (matriz[x*anchura + (y + 1)] == 0)
            {
                matriz[x*anchura + (y + 1)] = matriz[x*anchura + y];
                matriz[x*anchura + y] = 0;
            }
        }
    }
}

/*	mov_rightK
*	Global kernel function to move elements to the right
*/
__global__ void mov_rightK(int *matriz, int anchura, int altura)
{
    int x = threadIdx.x;
    int y = threadIdx.y;

    // Perform stacking and addition operations
    stack_right(matriz, anchura, altura, x, y);
    add_right(matriz, x, y, altura, anchura);
    __syncthreads();
    stack_right(matriz, anchura, altura, x, y);
}