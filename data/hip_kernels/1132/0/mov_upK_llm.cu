#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

void showMatriz(int *matriz, int anchura, int altura);
void generateSeeds(int *matriz, int ancho, int alto, int cantidad, char modo);
void gestionSemillas(int *matriz, int ancho, int numeroSemillas, int alto, char modo);
int checkFull(int *matriz, int tamano);
bool checkMove(int *matriz, int ancho, int alto);
void guardar(int vidas, int *tablero, int altura, int anchura, char dificultad);
int* cargar();
int* MostrarEspecificaciones();

hipError_t cudaStatus;

/*	add_up
*	Función del kernel para sumar hacia arriba todos los números que sean iguales.
*/
__device__ void stack_up(int *matriz, int anchura, int altura, int x, int y) {
    // Optimize loop condition using a single calculation
    for (int i = altura - 1; i > 0; i--) {
        if ((x != 0) && (matriz[x * anchura + y] != 0) && matriz[x * anchura + (y - anchura)] == 0) {
            matriz[x * anchura + (y - anchura)] = matriz[x * anchura + y];
            matriz[x * anchura + y] = 0;
        }
        __syncthreads(); // Ensure all threads reach this point before proceeding
    }
}

__device__ void add_up(int *matriz, int x, int y, int altura, int anchura) {
    if (x != 0 && y < anchura) {
        if (matriz[x * anchura + y] != 0) {
            if (matriz[x * anchura + y] == matriz[(x - 1) * anchura + y]) {
                int iguales = 1; // Initialize the count with 1
                // Loop unrolled for performance improvement
                for (int i = 1; i <= x; i++) {
                    if (matriz[x * anchura + y] != matriz[(x - i) * anchura + y]) break;
                    iguales++;
                }
                if (iguales % 2 == 0) {
                    matriz[(x - 1) * anchura + y] *= 2;
                    matriz[x * anchura + y] = 0;
                }
            } else if (matriz[(x - 1) * anchura + y] == 0) {
                matriz[(x - 1) * anchura + y] = matriz[x * anchura + y];
                matriz[x * anchura + y] = 0;
            }
        }
    }
}

__global__ void mov_upK(int *matriz, int anchura, int altura) {
    int x = threadIdx.x;
    int y = threadIdx.y;

    stack_up(matriz, anchura, altura, x, y);
    __syncthreads(); // Avoid unnecessary synchronizations
    add_up(matriz, x, y, altura, anchura);
    __syncthreads();
    stack_up(matriz, anchura, altura, x, y);
}