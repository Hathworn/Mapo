#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

void showMatriz(int *matriz, int anchura, int altura);
void generateSeeds(int *matriz, int ancho, int alto, int cantidad, char modo);
void gestionSemillas(int *matriz, int ancho, int numeroSemillas, int alto, char modo);
int checkFull(int *matriz, int tamano);
bool checkMove(int *matriz, int ancho, int alto);
void guardar(int vidas, int *tablero, int altura, int anchura, char dificultad);
int* cargar();
int* MostrarEspecificaciones();

hipError_t cudaStatus;

/* Optimize the stack_down function using a single loop */
__device__ void stack_down(int *matriz, int anchura, int altura, int x, int y) {
    // Loop from the second last row upwards to optimize memory access
    for (int i = altura - 2; i >= 0; i--) {
        if ((matriz[i*anchura + y] != 0) && matriz[(i+1)*anchura + y] == 0) {
            matriz[(i + 1)*anchura + y] = matriz[i*anchura + y];
            matriz[i*anchura + y] = 0;
        }
        __syncthreads();
    }
}

/* Optimize the add_down function to reduce unnecessary computations */
__device__ void add_down(int *matriz, int x, int y, int altura, int anchura) {
    if (x < altura - 1 && y < anchura) {
        if (matriz[x*anchura + y] != 0) {
            if (matriz[x*anchura + y] == matriz[(x + 1)*anchura + y]) {
                matriz[(x + 1)*anchura + y] *= 2;
                matriz[x*anchura + y] = 0;
            } else if (matriz[(x + 1)*anchura + y] == 0) {
                matriz[(x + 1)*anchura + y] = matriz[x*anchura + y];
                matriz[x*anchura + y] = 0;
            }
        }
    }
}

__global__ void mov_downK(int *matriz, int anchura, int altura) {
    int x = threadIdx.x;
    int y = threadIdx.y;

    stack_down(matriz, anchura, altura, x, y);
    __syncthreads();
    add_down(matriz, x, y, altura, anchura);
    __syncthreads();
    stack_down(matriz, anchura, altura, x, y);
}