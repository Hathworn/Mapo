#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

void showMatriz(int *matriz, int anchura, int altura);
void generateSeeds(int *matriz, int ancho, int alto, int cantidad, char modo);
void gestionSemillas(int *matriz, int ancho, int numeroSemillas, int alto, char modo);
int checkFull(int *matriz, int tamano);
bool checkMove(int *matriz, int ancho, int alto);
void guardar(int vidas, int *tablero, int altura, int anchura, char dificultad);
int* cargar();
int* MostrarEspecificaciones();

hipError_t cudaStatus;

__device__ void stack_left(int *matriz, int anchura, int altura, int x, int y) {
    // Loop optimization: boundary condition inside the loop
    if ((y != 0) && (matriz[x * anchura + y] != 0)) {
        for (int i = 0; i < anchura - 1; i++) {
            if (matriz[x * anchura + (y - 1)] == 0) {
                matriz[x * anchura + (y - 1)] = matriz[x * anchura + y];
                matriz[x * anchura + y] = 0;
            }
            __syncthreads(); // Maintain thread synchronization
        }
    }
}

__device__ void add_left(int *matriz, int x, int y, int altura, int anchura) {
    if (y != 0 && y < anchura) {
        if (matriz[x * anchura + y] != 0) {
            if (matriz[x * anchura + y] == matriz[x * anchura + (y - 1)]) {
                int iguales = 0;
                iguales++;
                for (int i = 1; i <= y; i++) {
                    if (matriz[x * anchura + y] == matriz[x * anchura + (y - i)]) {
                        iguales++;
                    } else {
                        break;
                    }
                }
                if (iguales % 2 == 0) {
                    matriz[x * anchura + (y - 1)] = matriz[x * anchura + (y - 1)] * 2;
                    matriz[x * anchura + y] = 0;
                }
            } else if (matriz[x * anchura + (y - 1)] == 0) {
                matriz[x * anchura + (y - 1)] = matriz[x * anchura + y];
                matriz[x * anchura + y] = 0;
            }
        }
    }
}

__global__ void mov_leftK(int *matriz, int anchura, int altura) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // Change to global indexing
    int y = blockIdx.y * blockDim.y + threadIdx.y; // Change to global indexing

    if (x < altura && y < anchura) { // Ensure threads are within bounds
        stack_left(matriz, anchura, altura, x, y);
        add_left(matriz, x, y, altura, anchura);
        __syncthreads();
        stack_left(matriz, anchura, altura, x, y);
    }
}