#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    float *feats = features + frame * ldf;

    // Apply lifter coefficients in parallel
    if (cepstral_lifter != 0.0f) {
        for (int c = thread_id; c < num_cols; c += blockDim.x) {  // Use blockDim.x for better readability
            feats[c] *= lifter_coeffs[c];  // Update feature in-place
        }
    }

    // Use __syncthreads to ensure all threads have completed lifter application
    __syncthreads();

    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        if (energy_floor > 0.0f && energy < log(energy_floor)) {
            energy = log(energy_floor);
        }
        feats[0] = energy;  // Apply energy at the start of the feature
    }
}