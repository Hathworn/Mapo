#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

#define ar 311
#define ac_br 312
#define bc 115

using namespace std;

__global__ void mat_mul(int *d_A, int *d_B, int *d_C, int rowA, int colA, int rowB, int colB, int rowC, int colC)
{
    // Calculate unique row and column for each thread
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds to ensure thread computes valid index
    if(row < rowC && col < colC)
    {
        int sum = 0; // Initialize sum for C[row][col]
        for(int i = 0; i < colA; i++)
        {
            sum += d_A[row * colA + i] * d_B[i * colB + col]; // Accumulate product
        }
        d_C[row * colC + col] = sum; // Store result in C
    }
}