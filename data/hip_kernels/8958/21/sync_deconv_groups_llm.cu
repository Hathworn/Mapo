#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized HIP kernel
__global__ void sync_deconv_groups() {
    // Use syncthreads to ensure all threads are synchronized
    __syncthreads();
}