#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sigmoidKernel(float* input, float* output, int edge) {
    int position = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread position
    if (position < edge) { // Ensure threads do not access out of bounds
        float x = input[position]; // Cache input value in a register
        output[position] = 1.0f / (1.0f + expf(-x)); // Utilize fast math function for exp
    }
}