#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil_1d(int *in, int *out)
{
    // Calculate the global index only once outside the loop
    int gindex = threadIdx.x + blockIdx.x * blockDim.x + RADIUS;

    int result = 0;

    // Use shared memory to reduce global memory access
    extern __shared__ int sharedMem[];

    // Load data into shared memory with offset
    sharedMem[threadIdx.x + RADIUS] = in[gindex];
    if (threadIdx.x < RADIUS) {
        sharedMem[threadIdx.x] = in[gindex - RADIUS];
        sharedMem[threadIdx.x + blockDim.x + RADIUS] = in[gindex + blockDim.x];
    }
    __syncthreads(); // Ensure all threads have loaded their data

    for (int offset = -RADIUS; offset <= RADIUS; offset++)
        result += sharedMem[threadIdx.x + RADIUS + offset]; // Access shared memory

    // Store the result
    out[gindex - RADIUS] = result;
}