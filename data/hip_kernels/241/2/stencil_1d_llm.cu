#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1d(int *in, int *out)
{
    // __shared__ keyword to declare variables in shared block memory
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + (blockIdx.x * blockDim.x) + RADIUS;
    int lindex = threadIdx.x + RADIUS;

    // Load main elements and halo in single statement for memory coalescing
    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS)
    {
        temp[lindex - RADIUS] = in[gindex - RADIUS];
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }

    __syncthreads();

    // Unroll the loop for performance
    int result = 0;
    result += temp[lindex - 1];
    result += temp[lindex];
    result += temp[lindex + 1];

    // Store the result
    out[gindex-RADIUS] = result;
}