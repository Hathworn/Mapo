#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil_2d(int *in, int *out) {
    // Calculate the global row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Define tile size and size of the apron
    const int TILE_WIDTH = blockDim.x;
    const int APERTURE = 1;  // Assuming a 3x3 stencil

    // Shared memory to hold the input tile and its halo
    __shared__ int tile[TILE_WIDTH + 2 * APERTURE][TILE_WIDTH + 2 * APERTURE];
    
    // Load data into shared memory, including the halo
    int haloRow = row - APERTURE;
    int haloCol = col - APERTURE;
    if (haloRow >= 0 && haloRow < gridDim.y * TILE_WIDTH + 2 * APERTURE &&
        haloCol >= 0 && haloCol < gridDim.x * TILE_WIDTH + 2 * APERTURE) {
        tile[threadIdx.y + APERTURE][threadIdx.x + APERTURE] = in[haloRow * (gridDim.x * TILE_WIDTH) + haloCol];
    }

    // Load halo region (left, right, top, bottom)
    if (threadIdx.x < APERTURE) {
        if (haloCol - APERTURE >= 0) {
            tile[threadIdx.y + APERTURE][threadIdx.x] = in[haloRow * (gridDim.x * TILE_WIDTH) + (haloCol - APERTURE)];
        }

        if (tile[threadIdx.y + APERTURE][threadIdx.x + TILE_WIDTH + APERTURE] < gridDim.x * TILE_WIDTH) {
            tile[threadIdx.y + APERTURE][threadIdx.x + TILE_WIDTH + APERTURE] = 
                in[haloRow * (gridDim.x * TILE_WIDTH) + (haloCol + TILE_WIDTH)];
        }
    }

    if (threadIdx.y < APERTURE) {
        if (haloRow - APERTURE >= 0) {
            tile[threadIdx.y][threadIdx.x + APERTURE] = in[(haloRow - APERTURE) * (gridDim.x * TILE_WIDTH) + haloCol];
        }

        if (haloRow + TILE_WIDTH + APERTURE < gridDim.y * TILE_WIDTH) {
            tile[threadIdx.y + TILE_WIDTH + APERTURE][threadIdx.x + APERTURE] = 
                in[(haloRow + TILE_WIDTH) * (gridDim.x * TILE_WIDTH) + haloCol];
        }
    }

    __syncthreads();

    // Apply the stencil operation
    if (threadIdx.x >= APERTURE && threadIdx.x < TILE_WIDTH + APERTURE && 
        threadIdx.y >= APERTURE && threadIdx.y < TILE_WIDTH + APERTURE) {
        int result = 0;
        for (int dy = -APERTURE; dy <= APERTURE; dy++) {
            for (int dx = -APERTURE; dx <= APERTURE; dx++) {
                result += tile[threadIdx.y + dy][threadIdx.x + dx];
            }
        }
        if (row < gridDim.y * TILE_WIDTH && col < gridDim.x * TILE_WIDTH) {
            out[row * (gridDim.x * TILE_WIDTH) + col] = result;
        }
    }
}