#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Use thread_stride and loop unrolling for performance improvement
    int thread_stride = blockDim.x;
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; i += thread_stride * BLOCK) {
            int index_base = size * (filter + n * b) + i;
            #pragma unroll 4
            for (int j = 0; j < BLOCK; j += thread_stride) {
                int index = p + j + index_base;
                if (p + j < size) {
                    sum += delta[index] * x_norm[index];
                }
            }
        }
    }
    part[p] = sum;
    __syncthreads();

    // Using atomicAdd to reduce contention
    if (p == 0) {
        float block_sum = 0;
        for (int i = 0; i < BLOCK; ++i) {
            block_sum += part[i];
        }
        atomicAdd(&scale_updates[filter], block_sum);
    }
}