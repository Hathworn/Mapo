#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate the global index for the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop to cover all elements
    for (; i < N; i += blockDim.x * gridDim.x)
    {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}