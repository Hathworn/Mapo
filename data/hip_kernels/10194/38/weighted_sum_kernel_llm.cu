#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate flattened thread index
    int i = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x;
    if (i < n) {
        float weighted_a = s[i] * a[i]; // Precompute weighted a
        float weighted_b = (b ? s[i] * b[i] : 0); // Precompute weighted b and handle b being null
        c[i] = weighted_a + (1 - s[i]) * weighted_b; // Combine results
    }
}