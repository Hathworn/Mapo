#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Optimize by combining multiple threads' work using the shared memory more effectively
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += delta[index];
        }
    }

    __syncthreads(); // Ensure all threads have completed their summation

    // Reduce using shared memory by summing all local results in thread 0
    if (id == 0) {
        float sum = 0;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        mean_delta[filter] = sum * (-1.0f / sqrt(variance[filter] + 0.000001f));
    }
}