#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate the unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit for out-of-bounds threads
    if (i >= N) return;

    // Precompute division and modulo to increase efficiency
    int in_s = i % spatial;
    int in_c = (i / spatial) % layers;
    int b = i / (spatial * layers);

    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;

    // Use ternary conditional for cleaner decision logic
    out[(forward ? i2 : i1)] = x[(forward ? i1 : i2)];
}