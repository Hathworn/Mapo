#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Improved grid indexing
    if (id >= size) return;

    // Use integer division and modulus to calculate indices.
    int current_id = id;
    int i = current_id % minw; 
    current_id /= minw;
    int j = current_id % minh; 
    current_id /= minh;
    int k = current_id % minc; 
    current_id /= minc;
    int b = current_id % batch;

    // Calculate output and add indices.
    int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
    int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));
    out[out_index] += add[add_index]; // Perform addition
}