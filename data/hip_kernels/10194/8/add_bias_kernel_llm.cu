#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockDim.x * (blockIdx.y * (gridDim.x * blockIdx.z) + blockIdx.x) + threadIdx.x; // Compute global thread offset
    if (offset < size * n)
    {
        int filter = blockIdx.y;
        int batch = blockIdx.z;
        output[offset] += biases[filter]; // Efficient indexing and addition
    }
}