#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Determine which channel and batch this thread is processing
    int batch_index = id / c;
    int channel_index = id % c;

    // Initialize sum for averaging
    float sum = 0.0f;
    int num_elements = w * h;

    // Compute input and output indices
    int out_index = channel_index + c * batch_index;
    int base_in_index = batch_index * c * num_elements + channel_index * num_elements;

    // Accumulate sum for the area
    for (int i = 0; i < num_elements; ++i) {
        sum += input[base_in_index + i];
    }

    // Store the average value in the output array
    output[out_index] = sum / num_elements;
}