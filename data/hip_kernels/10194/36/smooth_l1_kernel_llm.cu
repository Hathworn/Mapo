#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Optimize index calculation using blockDim.x for better readability and performance
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);  // Use standard library function for absolute value
        if (abs_val < 1.0f) {
            error[i] = diff * diff;
            delta[i] = diff;
        }
        else {
            error[i] = 2.0f * abs_val - 1.0f;  // Ensure consistent floating-point usage
            delta[i] = copysignf(1.0f, diff); // Use copysignf for sign determination
        }
    }
}