#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    extern __shared__ float local[];  // Allocate shared memory dynamically

    int id = threadIdx.x;
    int filter = blockIdx.x;

    local[id] = 0.0f;  // Initialize shared memory
    __syncthreads();   // Ensure initialization is complete before computation

    // Calculate strides to minimize redundant computations and optimize memory access
    int stride = blockDim.x * gridDim.x;
    int thread_index = filter * spatial + id;

    // Unroll the loop for better warp utilization
    for(int j = 0; j < batch; ++j){
        for(int i = id; i < spatial; i += stride){
            int index = j * spatial * filters + thread_index + i;
            if (i + id < spatial) {
                local[id] += delta[index] * (x[index] - mean[filter]);
            }
        }
    }
    __syncthreads();  // Ensure all thread computations are complete

    // Reduce the results within the block
    if (id == 0) {
        variance_delta[filter] = 0.0f;
        for (int i = 0; i < blockDim.x; ++i) {
            variance_delta[filter] += local[i];
        }
        variance_delta[filter] *= -0.5 * powf(variance[filter] + 1e-6f, -1.5f);
    }
}