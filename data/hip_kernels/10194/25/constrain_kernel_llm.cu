#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure calculations only occur within valid indices
    if(i < N) {
        // Apply constraint using fminf and fmaxf
        // Use one statement for concise calculation
        X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i * INCX]));
    }
}