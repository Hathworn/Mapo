#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Process elements in stride pattern for better memory access
    for (int index = i; index < N; index += blockDim.x * gridDim.x) {
        Y[index * INCY + OFFY] = X[index * INCX + OFFX];
    }
}