#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    float sum = 0.0f;
    float largest = -INFINITY;

    // Use parallel reduction to find the largest value
    for(int i = 0; i < n; ++i) {
        largest = fmaxf(input[i], largest);
    }

    // Calculate exponentials and sum in a single pass
    for(int i = 0; i < n; ++i) {
        float e = expf((input[i] - largest) / temp);
        sum += e;
        output[i] = e;
    }

    // Compute final output in a separate loop
    for(int i = 0; i < n; ++i) {
        output[i] /= sum;
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output)
{
    int b = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x + threadIdx.x;
    if(b >= batch) return;

    // Each thread computes softmax for a segment of input
    softmax_device(n, input + b * offset, temp, output + b * offset);
}