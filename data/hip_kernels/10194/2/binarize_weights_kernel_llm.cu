#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Use shared memory to reduce global memory access
    extern __shared__ float shared_weights[];

    float mean = 0;

    // Load weights into shared memory block by block
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[i] = weights[f * size + i];
        mean += abs(shared_weights[i]);
    }

    // Ensure all threads have completed their mean calculation
    __syncthreads();

    mean = mean / size;

    // Use shared memory to set binary weights
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (shared_weights[i] > 0) ? mean : -mean;
    }
}