#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use one-dimensional blockIdx layout
    int gridSize = blockDim.x * gridDim.x; // Calculate grid size

    while (i < N) { // Loop to handle more data than threads
        Y[i * INCY] = powf(X[i * INCX], ALPHA); // Use powf for single precision
        i += gridSize; // Increment index to next data chunk
    }
}