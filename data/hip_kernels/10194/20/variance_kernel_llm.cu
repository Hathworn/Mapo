#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Calculate scale outside the loop
    float scale = 1.0f / (batch * spatial - 1);

    // Flatten grid and block indices for 1D indexing
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit for threads out of range 
    if (i >= filters) return;

    // Initialize variance for current filter
    float var = 0.0f;
    
    // Unroll loops to improve performance
    #pragma unroll
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            float diff = x[index] - mean[i];
            var += diff * diff;
        }
    }

    // Store computed variance
    variance[i] = var * scale;
}