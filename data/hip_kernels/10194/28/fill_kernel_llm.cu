#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation
    int stride = gridDim.x * blockDim.x; // Compute stride based on grid dimensions
    while (i < N) {
        X[i * INCX] = ALPHA; // Assign ALPHA to array element
        i += stride; // Move to next stride
    }
}