#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate the global thread index.
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Only perform operations within bounds.
    if(i < n) {
        // Use temporary variable for intermediate result.
        float temp = a[i] * b[i];
        c[i] += temp;
    }
}