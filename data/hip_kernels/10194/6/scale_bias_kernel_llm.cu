#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int batch = blockIdx.z;
    int filter = blockIdx.y;
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int index = (batch * n + filter) * size + offset;
    
    // Check if index is within bounds and apply bias efficiently
    if(offset < size) 
    {
        output[index] *= biases[filter];
    }
}