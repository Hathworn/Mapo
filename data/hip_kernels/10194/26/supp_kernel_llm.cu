#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate thread index accounting for 1D grid and 1D block layout
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    // Ensure thread operates within vector bounds
    if(i < N) { 
        float val = X[i * INCX]; // Cache value to prevent repeated memory accesses
        if(val * val < ALPHA * ALPHA) 
            X[i * INCX] = 0; // Set value to 0 if condition met
    }
}