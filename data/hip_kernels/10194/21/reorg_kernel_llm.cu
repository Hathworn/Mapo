#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Precompute repetitive expressions to enhance performance.
    int in_w = i % w;
    int in_h = (i / w) % h;
    int in_c = ((i / w) / h) % c;
    int b = (((i / w) / h) / c) % batch;

    int out_c = c / (stride * stride);
    int offset = in_c / out_c;
    
    // Inline calculations for offset components.
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    int out_index = w2 + w * stride * (h2 + h * stride * (in_c % out_c + out_c * b));

    // Utilize conditional operator for concise assignment.
    if (forward) {
        out[out_index] = x[in_index];
    } else {
        out[in_index] = x[out_index];
    }
}