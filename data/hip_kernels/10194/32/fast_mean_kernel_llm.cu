#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    // Unroll the loop for increased performance
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial; i += threads){
            int index = j * spatial * filters + filter * spatial + i + id;
            local[id] += (i + id < spatial) ? x[index] : 0;
        }
    }

    // Use reduction pattern to compute sum
    __syncthreads();
    if(id < warpSize){
        for(int offset = warpSize; offset < threads; offset += warpSize){
            local[id] += local[id + offset];
        }
    }
    
    // Atomic operation to safely update mean
    if(id == 0){
        atomicAdd(&mean[filter], local[0] / (spatial * batch));
    }
}