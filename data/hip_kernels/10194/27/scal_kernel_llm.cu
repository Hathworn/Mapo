#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    int stride = gridDim.x * blockDim.x; // Calculate stride for grid-stride loop
    for (; i < N; i += stride) { // Grid-stride loop for better utilization
        X[i * INCX] *= ALPHA;
    }
}