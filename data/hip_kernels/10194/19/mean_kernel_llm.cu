#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Calculate scale outside of the kernel for efficient reuse
    extern __shared__ float shared_data[];
    
    float scale = 1.0 / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride to cover all filters
    for (; i < filters; i += blockDim.x * gridDim.x) {
        shared_data[threadIdx.x] = 0;

        // Accumulate sum in shared memory
        for (int j = 0; j < batch; ++j) {
            for (int k = 0; k < spatial; ++k) {
                int index = j * filters * spatial + i * spatial + k;
                shared_data[threadIdx.x] += x[index];
            }
        }

        // Reduce shared memory sums to a single value per thread block
        __syncthreads();
        if (threadIdx.x == 0) {
            float block_sum = 0;
            for (int t = 0; t < blockDim.x; ++t) {
                block_sum += shared_data[t];
            }
            mean[i] = block_sum * scale;
        }
    }
}