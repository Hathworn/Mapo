#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    // Compute a unique global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    // Initialize sum to 0
    float local_sum = 0.0f;

    // Accumulate values for each group in a local variable
    for (int k = 0; k < n; ++k) {
        local_sum += x[k * groups + i];
    }

    // Write the local sum to global memory
    sum[i] = local_sum;
}