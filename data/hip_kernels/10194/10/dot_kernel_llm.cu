#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;

    // Use shared memory to cache data
    extern __shared__ float shared[];
    float *shared_output = shared;

    for(int b = 0; b < batch; ++b){
        // Each block loads a batch of data into shared memory
        for(int i = threadIdx.x; i < size * n; i += blockDim.x) {
            shared_output[i] = output[b * size * n + i];
        }
        __syncthreads();

        // Reduced redundant calculations by using shared memory
        for(int i = 0; i < size; ++i){
            int i1 = f1 * size + i;
            int i2 = f2 * size + i;
            float val1 = shared_output[i1];
            float val2 = shared_output[i2];
            sum += val1 * val2;
            norm1 += val1 * val1;
            norm2 += val2 * val2;
        }
        __syncthreads();
    }
    
    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    sum = sum / norm;
    
    for(int b = 0; b < batch; ++b){
        for(int i = threadIdx.x; i < size; i += blockDim.x){
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            atomicAdd(&delta[i1], - scale * sum * shared_output[f2 * size + i] / norm);
            atomicAdd(&delta[i2], - scale * sum * shared_output[f1 * size + i] / norm);
        }
    }
}