#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global thread ID in 1D
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop to process more data if blocks are incomplete
    for (; i < N; i += blockDim.x * gridDim.x) {
        Y[i * INCY] *= X[i * INCX];
    }
}