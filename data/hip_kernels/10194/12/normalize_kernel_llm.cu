#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    // Precompute reciprocal of standard deviation to avoid recalculating it
    int f = (index / spatial) % filters;
    float inv_stddev = 1.0f / (sqrt(variance[f]) + .000001f);

    // Apply normalization
    x[index] = (x[index] - mean[f]) * inv_stddev;
}