#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_kernel(const float * __restrict__ x, const int n, float * __restrict__ binary)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride to handle large arrays
    int stride = blockDim.x * gridDim.x;

    // Loop over the entire array in strides
    for (; i < n; i += stride) {
        binary[i] = (x[i] >= 0) ? 1.0f : -1.0f;
    }
}