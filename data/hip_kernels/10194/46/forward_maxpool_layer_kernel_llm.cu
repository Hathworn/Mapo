#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes) {
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int c = in_c;
    
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;
    
    int out_index = id;
    int j = out_index % w;
    out_index /= w;
    int i = out_index % h;
    out_index /= h;
    int k = out_index % c;
    out_index /= c;
    int b = out_index;
    
    int w_offset = -pad;
    int h_offset = -pad;
    
    float max_val = -INFINITY;
    int max_index = -1;

    // Unroll the loop to improve performance
    for(int l = 0; l < size; l++) {
        for(int m = 0; m < size; m++) {
            int cur_h = h_offset + i * stride + l;
            int cur_w = w_offset + j * stride + m;
            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            int valid = (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w);
            float val = (valid != 0) ? input[index] : -INFINITY;

            // Use conditional assignment to reduce branching
            if (val > max_val) {
                max_val = val;
                max_index = index;
            }
        }
    }
    
    output[id] = max_val;
    indexes[id] = max_index;
}