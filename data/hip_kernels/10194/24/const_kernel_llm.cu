#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop unrolling to optimize memory accesses
    int gridSize = gridDim.x * blockDim.x;
    for (; i < N; i += gridSize)
    {
        X[i * INCX] = ALPHA;
    }
}