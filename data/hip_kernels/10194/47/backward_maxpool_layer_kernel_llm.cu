#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + 2*pad) / stride;
    int w = (in_w + 2*pad) / stride;
    int c = in_c;
    int area = (size - 1) / stride;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) return;

    int j = index % in_w;
    int i = (index / in_w) % in_h;
    int k = (index / (in_w * in_h)) % in_c;
    int b = index / (in_w * in_h * in_c);

    int w_offset = -pad;
    int h_offset = -pad;

    float d = 0;
    int j_start = max(0, ((j - w_offset) / stride) - area);
    int j_end = min(w - 1, ((j - w_offset) / stride) + area);
    int i_start = max(0, ((i - h_offset) / stride) - area);
    int i_end = min(h - 1, ((i - h_offset) / stride) + area);

    // Iterate over possible pooling window
    #pragma unroll
    for(int l = i_start; l <= i_end; ++l) {
        #pragma unroll
        for(int m = j_start; m <= j_end; ++m) {
            int out_index = m + w * (l + h * (k + c * b));
            d += (indexes[out_index] == index) ? delta[out_index] : 0;
        }
    }
    prev_delta[index] += d;
}