#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread operates within bounds and applies the mask
    if (i < n && mask[i] == mask_num) 
    {
        // Apply the mask
        x[i] = mask_num;
    }
}