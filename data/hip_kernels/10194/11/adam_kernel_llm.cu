#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    // Pre-compute shared values to reduce redundant calculations
    float one_minus_B1_t = 1.f - powf(B1, t);
    float one_minus_B2_t = 1.f - powf(B2, t);
    float sqrt_one_minus_B2_t = sqrtf(one_minus_B2_t);
    
    // Calculate flattened index within grid and block dimensions
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (index >= N) return;
    
    // Update x[index] with optimized calculation
    float m_val = m[index];
    float v_val = v[index];
    float sqrt_v_val_eps = sqrtf(v_val) + eps;
    
    x[index] -= (rate * sqrt_one_minus_B2_t / one_minus_B1_t) * m_val / sqrt_v_val_eps;
}