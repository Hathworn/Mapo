#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;
    int stride = gridDim.x * threads;  // Compute stride for coalesced memory access

    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += stride) {  // Use stride loop for efficiency
            int index = j * spatial * filters + filter * spatial + i;

            if (i < spatial) {
                local[id] += pow((x[index] - mean[filter]), 2);
            }
        }
    }

    __syncthreads();  // Ensure all threads have completed local summation

    if (id == 0) {
        float sum = 0;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        variance[filter] = sum / (spatial * batch - 1);
    }
}