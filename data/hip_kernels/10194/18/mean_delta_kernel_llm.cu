#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Use shared memory for mean_delta to avoid repeated global memory access
    __shared__ float shared_mean_delta;
    if (threadIdx.x == 0) {
        shared_mean_delta = 0.0f;
    }
    __syncthreads();

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            atomicAdd(&shared_mean_delta, delta[index]);
        }
    }

    if (threadIdx.x == 0) {
        shared_mean_delta *= (-1.0f / sqrt(variance[i] + .000001f));
        mean_delta[i] = shared_mean_delta;
    }
}