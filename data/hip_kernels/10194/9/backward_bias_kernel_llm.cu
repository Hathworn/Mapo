#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK 256  // Example value; adjust as appropriate for your use case

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Unroll the loop to improve memory access patterns
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index];
        }
    }

    part[p] = sum;
    __syncthreads();

    // Use a parallel reduction to accumulate partial sums
    for (int stride = BLOCK / 2; stride > 0; stride /= 2) {
        if (p < stride) {
            part[p] += part[p + stride];
        }
        __syncthreads();
    }

    // Only one thread needs to update the global memory
    if (p == 0) {
        atomicAdd(&bias_updates[filter], part[0]);
    }
}