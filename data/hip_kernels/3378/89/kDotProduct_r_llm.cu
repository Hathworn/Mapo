#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f; // Local sum variable to reduce shared memory access

    // Efficient loop with stride access pattern
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        sum += a[eidx] * b[eidx];
    }

    shmem[threadIdx.x] = sum; // Write to shared memory once

    __syncthreads();

    // Efficient reduction using warp-level operations
    if (threadIdx.x < 256) shmem[threadIdx.x] += shmem[threadIdx.x + 256];
    __syncthreads();
    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    __syncthreads();

    // Use warp shuffle operation for final reduction within a warp
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem; // Ensure visibility across warp
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
        if (threadIdx.x == 0) {
            target[blockIdx.x] = mysh[0];
        }
    }
}