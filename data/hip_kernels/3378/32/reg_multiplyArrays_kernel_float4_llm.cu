#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reg_multiplyArrays_kernel_float4(float4 *array1_d, float4 *array2_d)
{
    const int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    // Use built-in warp size for coalesced memory access 
    if (tid < c_VoxelNumber)
    {
        float4 a = array1_d[tid];
        float4 b = array2_d[tid];  // Correct array2_d in the multiplication
        array1_d[tid] = make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w);
    }
}