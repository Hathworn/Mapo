#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reg_multiplyArrays_kernel_float(float *array1_d, float *array2_d)
{
    // Calculate the global thread ID
    const int tid = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Optimize boundary check
    if(tid < c_VoxelNumber) {
        array1_d[tid] *= array2_d[tid];
    }
}