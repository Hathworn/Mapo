#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reg_addArrays_kernel_float4(float4 *array1_d, float4 *array2_d)
{
    // Calculate the thread index
    const int tid = blockIdx.x * blockDim.x * gridDim.y + blockIdx.y * blockDim.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if (tid < c_VoxelNumber) {
        // Load arrays into registers to optimize memory access
        float4 a = array1_d[tid];
        float4 b = array2_d[tid];
        
        // Perform the addition directly into array1_d to minimize register usage
        array1_d[tid] = make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
    }
}