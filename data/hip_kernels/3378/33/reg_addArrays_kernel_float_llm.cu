#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reg_addArrays_kernel_float(float *array1_d, float *array2_d)
{
    // Calculate linear thread index across the entire grid for improved indexing
    const int tid = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * (gridDim.x * blockDim.x);

    // Reduce branching with simple index boundary check
    if(tid < c_VoxelNumber)
    {
        array1_d[tid] += array2_d[tid];
    }
}