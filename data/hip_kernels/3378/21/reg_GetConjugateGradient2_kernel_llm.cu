#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reg_GetConjugateGradient2_kernel(float4 *nodeNMIGradientArray_d, float4 *conjugateG_d, float4 *conjugateH_d)
{
    // Calculate global thread ID
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure threads do not exceed array bounds
    if (tid < c_NodeNumber) {
        // Load node gradient
        float4 gradGValue = nodeNMIGradientArray_d[tid];
        
        // Compute G = -grad
        gradGValue = make_float4(-gradGValue.x, -gradGValue.y, -gradGValue.z, 0.0f);
        conjugateG_d[tid] = gradGValue;

        // Load and update H = G + scalingFactor * H
        float4 gradHValue = conjugateH_d[tid];
        gradHValue = make_float4(
            gradGValue.x + c_ScalingFactor * gradHValue.x,
            gradGValue.y + c_ScalingFactor * gradHValue.y,
            gradGValue.z + c_ScalingFactor * gradHValue.z,
            0.0f);
        conjugateH_d[tid] = gradHValue;
        
        // Update node gradient
        nodeNMIGradientArray_d[tid] = make_float4(-gradHValue.x, -gradHValue.y, -gradHValue.z, 0.0f);
    }
}