#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Optimize mean calculation by using shared memory
    extern __shared__ float shared_weights[]; // Shared memory allocation
    float sum = 0.0f;
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[i] = fabsf(weights[f*size + i]);
        sum += shared_weights[i];
    }
    __syncthreads();  // Ensure all threads have completed the sum

    // Reduce shared_weights to compute mean
    sum = 0.0f;
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        sum += shared_weights[i];
    }
    mean = sum / size;
    
    // Broadcast mean and binarize weights
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
    }
}