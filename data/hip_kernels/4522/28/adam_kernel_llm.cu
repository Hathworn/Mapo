#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation
    if (index >= N) return;

    // Pre-compute powers of B1 and B2 outside of the kernel if possible for efficiency
    float m_scaling = 1.f - powf(B1, t);
    float v_scaling = 1.f - powf(B2, t);

    float mhat = m[index] / m_scaling;  
    float vhat = v[index] / v_scaling;

    x[index] = x[index] + rate * mhat / (sqrtf(vhat) + eps);
}