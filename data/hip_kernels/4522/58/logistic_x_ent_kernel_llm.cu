#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate the unique thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Stride for grid-stride loop

    // Loop over the data with grid-stride to allow more blocks to be scheduled
    for (; i < n; i += stride) {
        float t = truth[i];
        float p = pred[i];
        error[i] = -t * log(p + 1e-7f) - (1 - t) * log(1 - p + 1e-7f); // Use a smaller epsilon
        delta[i] = t - p;
    }
}