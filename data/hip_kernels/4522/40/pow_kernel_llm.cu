#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;

    // Loop optimization for memory efficiency and better performance
    for (; i < N; i += gridSize) {
        Y[i * INCY] = pow(X[i * INCX], ALPHA);
    }
}