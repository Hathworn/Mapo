#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Calculate the global index once and reuse it
    int global_index = (batch * n + filter) * size + offset;

    // Perform boundary check and apply bias
    if(offset < size) output[global_index] *= biases[filter];
}