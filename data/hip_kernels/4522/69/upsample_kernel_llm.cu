#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if(index >= N) return;
    
    // Calculate indices with improved variable reuse and clarity
    int in_stride_w = w * h * c;
    int in_stride_h = w * h;
    int out_stride_w = w * stride;
    int out_stride_h = h * stride;
    
    int b = (index / (c * out_stride_h)) % batch;
    int out_c = (index / out_stride_h) % c;
    int out_h = (index / out_stride_w) % (h * stride);
    int out_w = index % (w * stride);
    
    int in_h = out_h / stride;
    int in_w = out_w / stride;
    
    int in_index = b * in_stride_w + out_c * in_stride_h + in_h * w + in_w;
    
    // Perform operation based on `forward` flag
    if(forward) {
        out[index] += scale * x[in_index];
    } else {
        atomicAdd(x + in_index, scale * out[index]);
    }
}