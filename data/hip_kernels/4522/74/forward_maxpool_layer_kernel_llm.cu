#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    // Calculate height and width of the output
    int h = (in_h + pad - size)/stride + 1;
    int w = (in_w + pad - size)/stride + 1;
    int c = in_c;

    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int j = (id / 1) % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    int w_offset = -pad/2;
    int h_offset = -pad/2;

    // Calculate the output index
    int out_index = j + w*(i + h*(k + c*b));
    float max = -INFINITY;
    int max_i = -1;

    // Reduce loop overhead by precomputing boundary checks once
    int start_h = max(0, h_offset + i * stride);
    int end_h = min(in_h, h_offset + i * stride + size);
    int start_w = max(0, w_offset + j * stride);
    int end_w = min(in_w, w_offset + j * stride + size);

    // Iterating through the pool window
    for(int l = start_h; l < end_h; ++l) {
        for(int m = start_w; m < end_w; ++m) {
            int index = m + in_w*(l + in_h*(k + b*in_c));
            float val = input[index];
            if(val > max) {
                max = val;
                max_i = index;
            }
        }
    }
    output[out_index] = max;
    indexes[out_index] = max_i;
}