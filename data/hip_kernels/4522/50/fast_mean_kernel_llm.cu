#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Optimizing loop by combining calculations within bounds
    for(int j = 0; j < batch; ++j) {
        for(int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += x[index];
        }
    }

    __syncthreads();

    // Use parallel reduction to compute mean
    if (id == 0) {
        float sum = 0;
        for (int i = 0; i < threads; i++) {
            sum += local[i];
        }
        mean[filter] = sum / (spatial * batch);
    }
}