#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation
    if (index >= N) return;

    int b = index / spatial;
    int i = index % spatial;
    float sum = 0;

    // Use shared memory for reduction
    extern __shared__ float shared_sum[];

    // Accumulate sum in shared memory
    for (int f = threadIdx.x; f < filters; f += blockDim.x) {
        int idx = b * filters * spatial + f * spatial + i;
        sum += x[idx] * x[idx];
    }

    shared_sum[threadIdx.x] = sum;
    __syncthreads();
    
    // Reduce sum within block
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Final reduction and normalization
    if (threadIdx.x == 0) {
        sum = sqrtf(shared_sum[0]);
        if (sum == 0) sum = 1;
        shared_sum[0] = sum;
    }
    __syncthreads();

    sum = shared_sum[0];

    // Normalize and calculate dx
    for (int f = threadIdx.x; f < filters; f += blockDim.x) {
        int idx = b * filters * spatial + f * spatial + i;
        x[idx] /= sum;
        dx[idx] = (1 - x[idx]) / sum;
    }
}