#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    // Calculate unique thread index using grid and block dimensions
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    
    // Ensure that we do not exceed the number of filters
    if (i >= filters) return;
    
    float sum = 0.0f; // Store the sum for each filter
    
    // Use a single loop to iterate over batch and spatial dimensions
    for (int j = 0; j < batch * spatial; ++j) {
        int index = (j / spatial) * filters * spatial + i * spatial + (j % spatial);
        sum += delta[index];
    }

    // Compute mean_delta using the precomputed sum
    mean_delta[i] = sum * (-1.f / sqrtf(variance[i] + .00001f));
}