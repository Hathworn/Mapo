#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    // Unroll the loop to improve memory access efficiency
    int stride = gridDim.x * gridDim.y * blockDim.x;
    for (int i = idx; i < N; i += stride)
    {
        X[i * INCX] = ALPHA;
    }
}