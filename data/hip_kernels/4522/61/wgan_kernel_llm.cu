#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void wgan_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation
    if (i < n) {
        float t = truth[i]; // Cache truth[i] for reuse
        error[i] = t ? -pred[i] : pred[i];
        delta[i] = (t > 0) ? 1 : -1;
    }
}