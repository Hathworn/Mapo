#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    if (id >= n) return;

    // Improved index calculation
    int k = id % c;
    int b = id / c;

    int out_index = k + c * b;
    float sum = 0.0f;

    // Use shared memory to reduce redundant calculations and improve memory access patterns
    extern __shared__ float shared_input[];
    
    for (int i = threadIdx.y; i < w * h; i += blockDim.y) {
        int in_index = i + h * w * (k + b * c);
        shared_input[threadIdx.y] = input[in_index];
        __syncthreads();  // Synchronize to ensure shared memory is updated
        sum += shared_input[threadIdx.y];
    }
    
    __syncthreads();  // Synchronize before output assignment
    output[out_index] = sum / (w * h);
}