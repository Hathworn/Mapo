#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation
    int gridSize = blockDim.x * gridDim.x;          // Calculate the stride for grid
    while (i < n) {                                 // Loop to cover all elements
        c[i] += a[i] * b[i];
        i += gridSize;                              // Increment by grid size
    }
}