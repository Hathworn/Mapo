#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by using shared memory for biases and loop tiling
    extern __shared__ float shared_biases[];
    int bias_offset = threadIdx.x;
    if (bias_offset < n) {
        shared_biases[bias_offset] = biases[bias_offset];
    }
    __syncthreads();
    
    int total_threads = blockDim.x * gridDim.x;

    for (int idx = index; idx < n * size * batch; idx += total_threads) {
        int i = idx % size;
        int j = (idx / size) % n;
        int k = idx / (size * n);

        // Use shared memory biases
        output[(k * n + j) * size + i] += shared_biases[j];
    }
}