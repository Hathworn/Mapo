#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.f/(batch * spatial);

    // Utilize blockIdx.x to compute the filter index directly.
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float sum = 0;
    
    // Combine loops to reduce computations
    int total = batch * spatial;
    for (int idx = 0; idx < total; ++idx) {
        int index = idx * filters + i;
        sum += x[index];
    }

    mean[i] = sum * scale;
}