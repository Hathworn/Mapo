#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global index based on the thread and block
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the computed index is within array bounds
    if (i < N) {
        // Perform element-wise multiplication
        Y[i * INCY] *= X[i * INCX];
    }
}