#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified grid and block index calculation
    if(id >= n) return;

    int b = id / c; // Compute batch index
    int k = id % c; // Compute channel index

    int out_index = k + c*b;
    float out_delta_val = out_delta[out_index] / (w*h); // Precompute division

    int offset = h*w*(k + b*c); // Precompute offset
    for(int i = 0; i < w*h; ++i){
        int in_index = i + offset;
        in_delta[in_index] += out_delta_val;
    }
}