#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[BLOCK];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0;

    int j, i, index;
    for(j = 0; j < batch; ++j) {
        // Ensure coalesced memory access for better performance
        for(i = id; i < spatial; i += threads) {
            index = j * spatial * filters + filter * spatial + i;
            local[id] += (i < spatial) ? delta[index] * (x[index] - mean[filter]) : 0;
        }
    }

    __syncthreads();

    // Reduce the results from all threads using atomic add for better performance
    if(id == 0) {
        float sum = 0;
        for(i = 0; i < threads; ++i) {
            sum += local[i];
        }
        atomicAdd(&variance_delta[filter], sum * -.5f * powf(variance[filter] + .00001f, -1.5f));
    }
}