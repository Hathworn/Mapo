#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    extern __shared__ float shared_mem[]; // use shared memory
    float *e_values = shared_mem;
    int thread_id = threadIdx.x;
    float largest = -INFINITY;

    // Calculate largest
    for (int i = thread_id; i < n; i += blockDim.x) {
        float val = input[i * stride];
        atomicMax(reinterpret_cast<int*>(&largest), __float_as_int(val));
    }
    __syncthreads();

    // Calculate e_values and sum
    float sum = 0;
    float current_val;
    for (int i = thread_id; i < n; i += blockDim.x) {
        current_val = expf((input[i * stride] - largest) / temp);
        e_values[i] = current_val;
        atomicAdd(&sum, current_val);
    }
    __syncthreads();

    // Normalize to get output
    for (int i = thread_id; i < n; i += blockDim.x) {
        output[i * stride] = e_values[i] / sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;

    int s = id % spatial;
    id = id / spatial;
    int g = id % groups;
    int b = id / groups;
    int goff = group_offset[g] * spatial;
    int boff = b * stride;

    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}