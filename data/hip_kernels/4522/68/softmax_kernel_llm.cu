#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    
    // Unroll loop for better performance
    #pragma unroll
    for (i = 0; i < n; ++i) {
        float val = input[i * stride]; // Correct type and use float
        largest = fmaxf(largest, val); // Use fmaxf for float comparison
    }
    
    #pragma unroll
    for (i = 0; i < n; ++i) {
        float e = expf(input[i * stride] / temp - largest / temp);
        sum += e;
        output[i * stride] = e;
    }
    
    float inv_sum = 1.0f / sum; // Calculate inverse of sum for efficiency
    #pragma unroll
    for (i = 0; i < n; ++i) {
        output[i * stride] *= inv_sum;
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread ID calculation
    if (id >= batch * groups) return;
    
    int b = id / groups;
    int g = id % groups;

    // Optimize memory access with offsets
    float *input_ptr = input + b * batch_offset + g * group_offset;
    float *output_ptr = output + b * batch_offset + g * group_offset;

    softmax_device(input_ptr, n, temp, stride, output_ptr);
}