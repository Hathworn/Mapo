#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    // Use shared memory for faster accumulation
    extern __shared__ float s_data[];

    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    // Initialize shared memory for this thread
    s_data[threadIdx.x] = 0.0f;

    // Loop over elements and accumulate using shared memory
    for (int k = threadIdx.x; k < n; k += blockDim.x) {
        s_data[threadIdx.x] += x[k * groups + i];
    }

    // Synchronize threads within the block
    __syncthreads();

    // Atomic addition to global sum for the final result
    atomicAdd(&sum[i], s_data[threadIdx.x]);
}