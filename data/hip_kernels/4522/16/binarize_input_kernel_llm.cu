#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = blockIdx.x * blockDim.x + threadIdx.x;
    if (s >= size) return;
    
    // Calculate mean once per thread and store it for re-use
    float mean = 0.0f;
    for (int i = 0; i < n; ++i) {
        mean += fabsf(input[i * size + s]);
    }
    mean = mean / n;
    
    // Use precomputed mean value for setting binary values
    float positiveMean = mean;
    float negativeMean = -mean;
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? positiveMean : negativeMean;
    }
}