#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the entire grid covers N elements and avoid unnecessary calculation
    int gridStride = blockDim.x * gridDim.x;
    
    for(; i < N; i += gridStride) {
        if((X[i*INCX] * X[i*INCX]) < (ALPHA * ALPHA)) {
            X[i*INCX] = 0;
        }
    }
}