#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate thread index using block and grid dimensions
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Ensure thread index is within bounds
    if (i * INCX < N) {
        X[i * INCX] = ALPHA;
    }
}