#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int c = in_c;
    int area = (size - 1) / stride;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int index = id;
    int j = index % in_w;
    index /= in_w;
    int i = index % in_h;
    index /= in_h;
    int k = index % in_c;
    index /= in_c;
    int b = index;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    float d = 0.0f;

    int out_w_base = (j - w_offset) / stride;
    int out_h_base = (i - h_offset) / stride;

    #pragma unroll // Unroll loops for improved performance
    for (int l = -area; l <= area; ++l) {
        int out_h = out_h_base + l;
        if (out_h < 0 || out_h >= h) continue;

        #pragma unroll
        for (int m = -area; m <= area; ++m) {
            int out_w = out_w_base + m;
            if (out_w < 0 || out_w >= w) continue;

            int out_index = out_w + w * (out_h + h * (k + c * b));
            if (indexes[out_index] == id) { // Check index match
                d += delta[out_index];
            }
        }
    }

    prev_delta[id] += d;
}