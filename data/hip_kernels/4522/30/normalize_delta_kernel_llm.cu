#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Flattened 1D index calculation
    if (index >= N) return;

    int f = (index / spatial) % filters;

    // Precompute common terms to optimize performance
    float inv_sqrt_variance = 1.f / sqrtf(variance[f] + .00001f);
    float variance_factor = variance_delta[f] * 2.f / (spatial * batch);
    float mean_factor = mean_delta[f] / (spatial * batch);

    // Combine all contributions to delta[index]
    delta[index] = delta[index] * inv_sqrt_variance + variance_factor * (x[index] - mean[f]) + mean_factor;
}