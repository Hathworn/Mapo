#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Use shared memory for better performance if multiple threads access the same data
    extern __shared__ float shared_mem[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < n){
        float pred_val = pred[i];  // Reduce global memory access by storing in a register
        float truth_val = truth[i]; // Reduce global memory access by storing in a register
        float diff = truth_val - pred_val;
        error[i] = diff * diff;
        delta[i] = diff;
    }
}