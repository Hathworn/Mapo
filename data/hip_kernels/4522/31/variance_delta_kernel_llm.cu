#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Initialize variance_delta array
    float var_delta_sum = 0;
    
    // Reduce repetitive access by pre-calculating multiplier
    float multiplier = -.5f * powf(variance[i] + .00001f, (float)(-3.f/2.f));

    // Iterate over batch and spatial dimensions
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            // Accumulate variance delta
            var_delta_sum += delta[index] * (x[index] - mean[i]);
        }
    }

    // Assign calculated value to variance_delta
    variance_delta[i] = var_delta_sum * multiplier;
}