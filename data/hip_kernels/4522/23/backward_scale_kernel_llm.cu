#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;
    // Use warp reduction to optimize sum computation within a block
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; i += BLOCK) {
            int index = p + i + size * (filter + n * b);
            if (p + i < size) sum += delta[index] * x_norm[index];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Use shared memory and partial reduction
    if (p < BLOCK / 2) {
        part[p] += part[p + BLOCK / 2];
    }
    __syncthreads();
    
    if (p == 0) {
        for (int i = 0; i < BLOCK / 2; ++i) scale_updates[filter] += part[i];
    }
}