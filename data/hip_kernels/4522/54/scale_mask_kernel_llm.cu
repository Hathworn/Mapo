#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize grid calculation assuming 1D block and grid
    int stride = gridDim.x * blockDim.x; // Compute stride for further iteration

    for (; i < n; i += stride) { // Loop through elements with stride
        if (mask[i] == mask_num) { // Only branch if necessary
            x[i] *= scale;
        }
    }
}