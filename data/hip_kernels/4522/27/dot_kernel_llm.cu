#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Calculate a single thread index
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    // Initialize accumulator variables
    float sum = 0.0f, norm1 = 0.0f, norm2 = 0.0f;
    int total_size = size * n;
    
    // Unroll loops for better performance
    for (int b = 0; b < batch; ++b) {
        int base_index1 = b * total_size + f1 * size;
        int base_index2 = b * total_size + f2 * size;
        for (int i = 0; i < size; ++i) {
            int i1 = base_index1 + i;
            int i2 = base_index2 + i;
            float output_i1 = output[i1];
            float output_i2 = output[i2];
            sum += output_i1 * output_i2;
            norm1 += output_i1 * output_i1;
            norm2 += output_i2 * output_i2;
        }
    }
    
    // Precompute norms
    float inv_norm1 = rsqrtf(norm1);
    float inv_norm2 = rsqrtf(norm2);
    float norm = inv_norm1 * inv_norm2;
    sum *= norm;

    // Update delta with optimized access
    for (int b = 0; b < batch; ++b) {
        int base_index1 = b * total_size + f1 * size;
        int base_index2 = b * total_size + f2 * size;
        for (int i = 0; i < size; ++i) {
            int i1 = base_index1 + i;
            int i2 = base_index2 + i;
            float output_i1 = output[i1];
            float output_i2 = output[i2];
            float scaled_sum = -scale * sum;
            atomicAdd(&delta[i1], scaled_sum * output_i2 / norm);
            atomicAdd(&delta[i2], scaled_sum * output_i1 / norm);
        }
    }
}