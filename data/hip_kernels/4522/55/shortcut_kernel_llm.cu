#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) return;

    // Use integer math to calculate indices to avoid divisions
    int temp = id;
    int i = temp % minw;
    temp /= minw;
    int j = temp % minh;
    temp /= minh;
    int k = temp % minc;
    temp /= minc;
    int b = temp % batch;

    int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
    int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));

    // Minimize memory reads by caching computed values
    float out_val = out[out_index];
    float add_val = add[add_index];

    // Calculate the result directly into the output
    out[out_index] = s1 * out_val + s2 * add_val;
}