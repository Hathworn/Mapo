#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Calculate global index for thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a single conditional check for valid index
    if (i < n) {
        binary[i] = __float2int_rn(x[i] >= 0) * 2 - 1; // Ternary condition simplified
    }
}