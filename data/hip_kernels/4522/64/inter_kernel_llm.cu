#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate global thread index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x 
            + blockIdx.y * gridDim.x * blockDim.x;

    // Use local variables for calculations to reduce memory access
    const int totalXY = NX + NY;

    if (i < totalXY * B) {
        int b = i / totalXY;
        int j = i % totalXY;

        // Optimize conditional branch
        OUT[i] = (j < NX) ? X[b * NX + j] : Y[b * NY + j - NX];
    }
}