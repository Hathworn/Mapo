#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    float h = hsv.x;
    float s = hsv.y;
    float v = hsv.z;
    float r, g, b;

    if (s == 0) {
        r = g = b = v; // If saturation is zero, the color is gray
    } else {
        int index = (int) floorf(h);
        float f = h - index;
        float p = v * (1 - s);
        float q = v * (1 - s * f);
        float t = v * (1 - s * (1 - f));

        // Use a more optimized switch-case construct
        switch (index) {
            case 0:  r = v; g = t; b = p; break;
            case 1:  r = q; g = v; b = p; break;
            case 2:  r = p; g = v; b = t; break;
            case 3:  r = p; g = q; b = v; break;
            case 4:  r = t; g = p; b = v; break;
            default: r = v; g = p; b = q; break;
        }
    }
    return make_float3(fminf(fmaxf(r, 0.0f), 1.0f),
                       fminf(fmaxf(g, 0.0f), 1.0f),
                       fminf(fmaxf(b, 0.0f), 1.0f)); // Clamp values between 0 and 1
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    float r = rgb.x;
    float g = rgb.y;
    float b = rgb.z;

    float max = fmaxf(r, fmaxf(g, b));
    float min = fminf(r, fminf(g, b));
    float delta = max - min;

    float h, s, v = max;

    s = (max == 0) ? 0 : delta / max; // Avoid division by zero for saturation

    if (max == min) {
        h = 0; // Achromatic case
    } else if (r == max) {
        h = (g - b) / delta;
    } else if (g == max) {
        h = 2 + (b - r) / delta;
    } else {
        h = 4 + (r - g) / delta;
    }
    if (h < 0) h += 6;

    return make_float3(h, s, v);
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int size = batch * w * h;
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    
    int x = id % w;
    int y = (id / w) % h; // Calculate y directly
    
    int imgId = id / (w * h); // Extract the image index
    size_t offset = imgId * h * w * 3; // Calculate offset using imgId

    float rshift = rand[0];
    float gshift = rand[1];
    float bshift = rand[2];
    float r0 = rand[8 * imgId + 0];
    float r1 = rand[8 * imgId + 1];
    float r2 = rand[8 * imgId + 2];
    float r3 = rand[8 * imgId + 3];

    saturation = r0 * (saturation - 1) + 1;
    saturation = (r1 > .5f) ? 1.f / saturation : saturation;
    exposure = r2 * (exposure - 1) + 1;
    exposure = (r3 > .5f) ? 1.f / exposure : exposure;

    image += offset;
    float r = image[x + w * (y + h * 0)];
    float g = image[x + w * (y + h * 1)];
    float b = image[x + w * (y + h * 2)];
    float3 rgb = make_float3(r, g, b);

    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }

    image[x + w * (y + h * 0)] = rgb.x * scale + translate + (rshift - .5f) * shift;
    image[x + w * (y + h * 1)] = rgb.y * scale + translate + (gshift - .5f) * shift;
    image[x + w * (y + h * 2)] = rgb.z * scale + translate + (bshift - .5f) * shift;
}