#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate linear thread index
    int stride = blockDim.x * gridDim.x; // Calculate the stride for iterations
    int total_elements = (NX + NY) * B; // Total number of elements

    for (int i = idx; i < total_elements; i += stride) { // Loop over elements with stride
        int b = i / (NX + NY); // Batch index
        int j = i % (NX + NY); // Element index within a batch
        if (j < NX) {
            if (X) X[b * NX + j] += OUT[i]; // Process X array
        } else {
            if (Y) Y[b * NY + j - NX] += OUT[i]; // Process Y array
        }
    }
}