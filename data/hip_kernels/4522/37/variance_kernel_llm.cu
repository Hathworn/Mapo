#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Calculate scale once and move it outside kernel loop
    float scale = 1.f / (batch * spatial - 1);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Bound check for filter range
    if (i >= filters) return;
    
    // Use local variables for variance computation
    float local_variance = 0.0;

    for (int j = 0; j < batch; ++j)
    {
        for (int k = 0; k < spatial; ++k)
        {
            int index = j * filters * spatial + i * spatial + k;
            local_variance += powf((x[index] - mean[i]), 2);
        }
    }

    // Update global variance array with computed variance
    variance[i] = local_variance * scale;
}