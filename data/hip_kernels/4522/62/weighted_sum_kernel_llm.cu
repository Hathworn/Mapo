#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid-stride loop for efficient memory access
    for (; i < n; i += blockDim.x * gridDim.x)
    {
        // Perform weighted sum calculation
        c[i] = s[i] * a[i] + (1 - s[i]) * (b ? b[i] : 0);
    }
}