#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified grid index calculation
    if(i < n){
        float t = truth[i];
        float p = pred[i];
        // Use math function with HIP prefix for compatibility and performance
        error[i] = (t) ? -__logf(p) : 0.0f;
        delta[i] = t - p;
    }
}