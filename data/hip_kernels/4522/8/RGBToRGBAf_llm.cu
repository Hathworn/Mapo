#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RGBToRGBAf(uchar3* srcImage, float4* dstImage, int width, int height)
{
    // Calculate pixel position
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Early exit for out-of-bound threads
    if( x >= width || y >= height )
        return;

    int pixel = y * width + x;
    const float s = 1.0f;
    const uchar3 px = srcImage[pixel];
    
    // Directly assign values with scale
    dstImage[pixel] = make_float4(px.x * s, px.y * s, px.z * s, 255.0f * s);
}