#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    extern __shared__ float local[]; // Use dynamic shared memory

    int id = threadIdx.x;
    int filter = blockIdx.x;
    int total_spatial = batch * spatial; // Calculate total once

    local[id] = 0.0f;

    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += blockDim.x) { // Optimize loop to avoid index calculations
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += powf(x[index] - mean[filter], 2);
        }
    }

    __syncthreads();

    if (id == 0) {
        float sum = 0.0f;
        for (int i = 0; i < blockDim.x; ++i) {
            sum += local[i];
        }
        variance[filter] = sum / (total_spatial - 1); // Use total_spatial for readability
    }
}