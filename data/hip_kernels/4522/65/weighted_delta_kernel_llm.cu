#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid stride loop for optimization
    for (; i < n; i += blockDim.x * gridDim.x) {
        if (da) da[i] += dc[i] * s[i];
        if (db) db[i] += dc[i] * (1 - s[i]);
        ds[i] += dc[i] * (a[i] - b[i]);
    }
}