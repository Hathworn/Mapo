#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0.0f;

    // Loop unrolling for improved performance
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; i += BLOCK) {
            int index = p + i + size * (filter + n * b);
            if (p + i < size) {
                sum += delta[index];
            }
        }
    }
    part[p] = sum;

    // An optimized reduction using shared memory
    __syncthreads();
    for (int offset = BLOCK / 2; offset > 0; offset >>= 1) {
        if (p < offset) {
            part[p] += part[p + offset];
        }
        __syncthreads();
    }

    // Only one thread writes the result
    if (p == 0) {
        atomicAdd(&bias_updates[filter], part[0]);
    }
}