#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    // Use shared memory to optimize accumulation
    extern __shared__ float shared_sum[];
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= n) return;

    float sum = 0.0f;
    for (int b = 0; b < batch; ++b) {
        int i = b * n + index;
        sum += delta[i];
    }
    shared_sum[threadIdx.x] = sum;

    // Synchronize threads
    __syncthreads();

    // Reduce sum in shared memory
    if (threadIdx.x == 0) {
        float block_sum = 0.0f;
        for (int i = 0; i < blockDim.x; ++i) {
            block_sum += shared_sum[i];
        }
        atomicAdd(&bias_updates[blockIdx.x + blockIdx.y * gridDim.x], block_sum);
    }
}