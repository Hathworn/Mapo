#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure all threads operate within bounds
    for (; i < N; i += blockDim.x * gridDim.x) {
        X[i * INCX] *= ALPHA;
    }
}