#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index considering grid and block dimensions
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Ensure the index is within bounds and apply constraints
    if(i < N) X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i * INCX]));
}