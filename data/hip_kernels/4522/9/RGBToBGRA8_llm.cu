#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RGBToBGRA8(float3* srcImage, uchar4* dstImage, int width, int height, float scaling_factor)
{
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    if (x < width && y < height)  // Single boundary check
    {
        const int pixel = y * width + x;
        const float3 px = srcImage[pixel];

        // Direct assignment with make_uchar4
        dstImage[pixel] = make_uchar4(
            __fmul_rn(px.z, scaling_factor),
            __fmul_rn(px.y, scaling_factor),
            __fmul_rn(px.x, scaling_factor),
            255.0f * scaling_factor);
    }
}