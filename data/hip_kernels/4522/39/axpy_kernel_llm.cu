#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Combine offsets into pointers to remove repetitive calculations
    float *x_ptr = X + OFFX;
    float *y_ptr = Y + OFFY;
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure restricted range for efficient memory access
    if (i < N) {
        y_ptr[i * INCY] += ALPHA * x_ptr[i * INCX];
    }
}