#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[BLOCK];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Use shared memory to reduce memory access latency
    for(int j = 0; j < batch; ++j){
        int base_idx = j * spatial * filters + filter * spatial;
        for(int i = id; i < spatial; i += threads){
            local[id] += delta[base_idx + i];
        }
    }

    __syncthreads();

    // Perform reduction using parallel reduction
    if(id < 32) {
        #pragma unroll
        for(int offset = 32; offset > 0; offset >>= 1) {
            local[id] += __shfl_down_sync(0xFFFFFFFF, local[id], offset);
        }
    }

    if(id == 0){
        mean_delta[filter] = local[0] * (-1.f / sqrtf(variance[filter] + .00001f));
    }
}
```
