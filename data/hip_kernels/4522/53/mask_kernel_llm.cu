#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    // Calculate global thread index using block and grid dimensions
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * (gridDim.x * blockDim.x); 

    // Ensure thread index is within array bounds and perform masking
    if (i < n) 
    {
        // Use mask for conditional assignment
        if (mask[i] == mask_num) 
        {
            x[i] = val;
        }
    }
}