#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int j = id % w;         // Calculate w index
    int bc_h = id / w;      // Pre-calculate block and height index
    int i = bc_h % h;       // Extract h index
    int kc_b = bc_h / h;    // Pre-calculate channel and batch index
    int k = kc_b % c;       // Extract channel index
    int b = kc_b / c;       // Extract batch index

    int out_index = j + w * (i + h * (k + b * c));
    int w_offset = -(size / 2);
    int h_offset = -(size / 2);

    float delta_val = 0; // Accumulator for delta update
    for (int l = 0; l < size; ++l) {
        int cur_h = h_offset + i + l;
        if (cur_h >= 0 && cur_h < h) { // Check valid height once per row
            for (int m = 0; m < size; ++m) {
                int cur_w = w_offset + j + m;
                if (cur_w >= 0 && cur_w < w) { // Check valid width
                    int index = cur_w + w * (cur_h + h * (k + b * c));
                    delta_val += rate * (x[index] - x[out_index]);
                }
            }
        }
    }
    delta[out_index] += delta_val; // Update delta once per thread
}