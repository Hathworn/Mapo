#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified 1D grid mapping
    int stride = blockDim.x * gridDim.x;           // Calculate stride for 1D grid

    // Loop over elements with stride pattern to ensure all elements are processed in larger grids
    for (int index = i; index < N; index += stride) {
        X[index * INCX] += ALPHA;
    }
}