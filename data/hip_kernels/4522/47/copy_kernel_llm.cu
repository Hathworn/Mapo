#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize: Use grid-stride loop to allow the kernel to process more elements per thread
    for (int index = i; index < N; index += blockDim.x * gridDim.x)
    {
        Y[index * INCY + OFFY] = X[index * INCX + OFFX];
    }
}