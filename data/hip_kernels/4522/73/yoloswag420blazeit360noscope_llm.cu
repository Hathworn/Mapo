#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplify id calculation for 1D grid
    if(id < size) { // Use braces for better readability
        float randValue = rand[id]; // Cache rand[id] to reduce global memory access
        if(randValue >= prob) { // Compare once and scale if necessary
            input[id] *= scale;
        } else {
            input[id] = 0.0f; // Explicitly assign zero to improve clarity
        }
    }
}