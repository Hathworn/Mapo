#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define CUDA_KERNEL_LOOP(i, n) \
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;

// Optimized kernel function
__global__ void calculate_dbias_kernel(int n, const float* grad_output, float* grad_bias, const int out_channels, const int height_out, const int width_out) {
    // Calculate index using thread and block information
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Loop with stride to cover all elements
    for (int i = index; i < n; i += blockDim.x * gridDim.x) {
        int c_col = (i / width_out / height_out) % out_channels;
        float value = grad_output[i];
        atomicAdd(&grad_bias[c_col], value); // Safely accumulate into grad_bias with atomic operation
    }
}