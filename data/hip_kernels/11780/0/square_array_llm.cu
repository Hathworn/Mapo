#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    // Use shared memory for faster access
    extern __shared__ float shared_a[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load data into shared memory
    if (idx < N) shared_a[threadIdx.x] = a[idx];
    __syncthreads();

    // Compute square in shared memory
    if (idx < N) shared_a[threadIdx.x] *= shared_a[threadIdx.x];
    __syncthreads();

    // Write results back to global memory
    if (idx < N) a[idx] = shared_a[threadIdx.x];
}