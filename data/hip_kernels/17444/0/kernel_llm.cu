#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *w_vect, float *train, float *partition, int rows, int cols){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < rows) { // Ensure thread only works on valid data
        float temp = 0.0f;
        for (int i = 0; i < cols; i++){
            temp += w_vect[i] * train[i * rows + tid];
        }
        partition[tid] = temp;
    }
}