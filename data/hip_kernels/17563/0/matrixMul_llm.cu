#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMul(int* A, int* B, int* C, int aF, int aC, int bF, int bC, int cF, int cC) {
    // Compute each thread's global row and column index
    int row = (blockIdx.y * blockDim.y) + threadIdx.y;
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (aC != bF) return; // Early exit if dimensions don't match

    // Ensure threads operate within bounds
    if ((row < aF) && (col < bC)) {
        int sum = 0; // Use a local variable to accumulate results
        for (int k = 0; k < aC; ++k) {
            // Perform and accumulate multiplication
            sum += A[row * aC + k] * B[k * bC + col];
        }
        C[row * cC + col] = sum; // Write result to global memory
    }
}