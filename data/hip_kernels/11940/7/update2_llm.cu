#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update2(float *alphaMinusBeta_out, const float *rho, const float *yDotZ, const float *alpha)
{
    // Load inputs from global memory to local variables for quick access
    float rho_val = *rho;
    float yDotZ_val = *yDotZ;
    float alpha_val = *alpha;

    // Compute beta using local variables
    float beta = rho_val * yDotZ_val;

    // Perform subtraction and store the result
    *alphaMinusBeta_out = alpha_val - beta;
}