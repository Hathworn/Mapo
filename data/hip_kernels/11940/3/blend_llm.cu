#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blend(float *cmap, float* oldd, float* newd, float weight, int * params)
{
    int ax = blockIdx.x * blockDim.x + threadIdx.x;
    int ay = blockIdx.y * blockDim.y + threadIdx.y;
    
    int ch = params[0];
    int ah = params[1];
    int aw = params[2];
    
    int slice_a = ah * aw;
    int pitch_a = aw;
    
    float thre = 0.05;
    
    if (ax < aw && ay < ah)
    {
        float fa = cmap[ay * pitch_a + ax];
        // Use ternary for fa calculation, avoiding branching
        fa = (fa < thre) ? 0.0f : weight;
        
        for (int i = 0; i < ch; i++)
        {
            int idx = i * slice_a + ay * pitch_a + ax;
            // Precompute indices to reduce repetitive calculations
            // Blend old and new data with precomputed index
            newd[idx] = oldd[idx] * fa + newd[idx] * (1.0 - fa);
        }
    }
}