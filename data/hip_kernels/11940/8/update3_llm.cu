#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update3(float *rho_out, float *H0_out, const float *yDotS, const float *yDotY)
{
    // Read values into registers to reduce global memory accesses
    float yDotS_val = *yDotS;
    float yDotY_val = *yDotY;

    // Compute rho_out using the value stored in the register
    *rho_out = 1.0f / yDotS_val;

    // Check the condition and compute H0_out only when necessary
    if (yDotY_val > 1e-5)
    {
        *H0_out = yDotS_val / yDotY_val;
    }
}