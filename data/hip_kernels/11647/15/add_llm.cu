#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(long* in, long* out, int offset, int n) {
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid < n) { // Check gid once to improve efficiency
        long temp = in[gid];
        if (gid >= offset) 
            temp += in[gid - offset];
        out[gid] = temp;
    }
}