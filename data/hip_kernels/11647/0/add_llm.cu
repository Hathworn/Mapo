#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 10000000 //input data size: 10,000,000
#define BLOCKSIZE 1024

/* prefix sum */

using namespace std;

__global__ void add(double* in, double* out, int offset, int n) {

    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Check bounds before accessing memory to ensure thread safety
    if(gid < n) {
        // Sequential memory access; may improve coalescing
        double temp = in[gid];
        if(gid >= offset)
            temp += in[gid - offset];
        out[gid] = temp;
    }
}