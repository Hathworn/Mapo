#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(double* in, double* out, int offset, int n){
    int gid = blockIdx.x * blockDim.x + threadIdx.x; // Rearrange to improve readability

    // Check bounds once to avoid accessing out of bounds
    if(gid < n) {
        out[gid] = in[gid]; // Copy input element to output

        // Perform addition only if within valid range
        if(gid >= offset) {
            out[gid] += in[gid - offset];
        }
    }
}