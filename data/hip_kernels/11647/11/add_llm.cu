#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(double* in, double* out, int offset, int n) {
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    // Early exit to prevent unnecessary computation
    if (gid < n) {
        out[gid] = in[gid];
        // Perform addition only when necessary
        if (gid >= offset) {
            out[gid] += in[gid - offset];
        }
    }
}