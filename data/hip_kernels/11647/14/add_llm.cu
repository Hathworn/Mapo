#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(double* in, double* out, int offset, int n) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure gid is within bounds and offset logic is applied efficiently
    if (gid < n) {
        double temp = in[gid];
        if (gid >= offset) {
            temp += in[gid - offset];
        }
        out[gid] = temp;
    }
}