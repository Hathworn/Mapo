#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(double* out, int n){
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if gid is in valid range before writing
    if (gid < n) {
        out[gid] = 0.0;
    }
}