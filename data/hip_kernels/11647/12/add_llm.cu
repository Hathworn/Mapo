#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int* in, int* out, int n) {

    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= n) return;
    
    extern __shared__ int temp[];

    // Store initial values in shared memory
    temp[threadIdx.x] = (gid > 0) ? in[gid - 1] : 0;
    __syncthreads();

    // Use single index for temp memory to reduce computation
    for (int offset = 1; offset < n; offset <<= 1) {
        int val = (threadIdx.x >= offset) ? temp[threadIdx.x - offset] : 0;
        __syncthreads();
        temp[threadIdx.x] += val;
        __syncthreads();
    }
    out[gid] = temp[threadIdx.x];
}