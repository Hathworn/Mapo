#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(long* in, long* out, int offset, int n) {
    // Calculate the global thread ID:
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    // Early exit for threads with ID >= n, improving efficiency:
    if (gid >= n) return;
    // Perform the addition operations with bounds check:
    long temp = in[gid];
    if (gid >= offset) {
        temp += in[gid - offset];
    }
    out[gid] = temp;
}