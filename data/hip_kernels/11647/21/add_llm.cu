#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(double* __restrict__ in, double* __restrict__ out, int offset, int n) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds only once
    if (gid >= n) return;

    out[gid] = in[gid];

    if (gid >= offset) {
        out[gid] += in[gid - offset];
    }
}