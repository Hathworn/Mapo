#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int* in, int* out, int offset, int n){
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    // Check if index is within bounds
    if (gid < n) {
        out[gid] = in[gid];
        // Perform addition only if gid is greater than or equal to offset
        if (gid >= offset) {
            out[gid] += in[gid-offset];
        }
    }
}