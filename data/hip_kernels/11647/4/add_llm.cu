#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int* in, int* out, int offset, int n) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global ID
    if (gid < n) { // Check bounds
        int value = in[gid]; // Load value from input
        if (gid >= offset) {
            value += in[gid - offset]; // Add offset value if valid
        }
        out[gid] = value; // Write result to output
    }
}