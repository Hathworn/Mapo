#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_cuda(int* d_a, int* d_b, int* d_c, int m, int n) {

    // Get row and column indices
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Row i of matrix C
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Column j of matrix C

    // Check bounds to avoid out-of-bounds access
    if (i < m && j < m) {
        int cell = 0;
        for (int k = 0; k < n; k++) {
            cell += d_a[i * n + k] * d_b[k * m + j]; // Accumulate product of row and column
        }
        d_c[i * m + j] = cell; // Store result in matrix C
    }
}