#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int* in, int d, int n) {
    // Calculate global thread ID
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if gid is within bounds
    if (gid < n) {
        // Calculate the offset based on d
        int pre = (d == 0) ? 1 : (1 << d);
        
        // Update array element conditionally
        if (gid >= pre) {
            in[gid] += in[gid - pre];
        }
    }
}