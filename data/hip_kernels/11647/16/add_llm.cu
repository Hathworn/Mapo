#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int* in, int* out, int n) {

    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= n) return;

    extern __shared__ int temp[];

    // Load input into shared memory
    temp[threadIdx.x] = in[gid];
    __syncthreads();

    for (int offset = 1; offset < n; offset <<= 1) {
        int val = 0;

        // Read data from shared memory and perform the addition
        if (threadIdx.x >= offset) {
            val = temp[threadIdx.x - offset];
        } else if (gid >= offset) {
            val = in[gid - offset];
        }
        
        __syncthreads();
        
        // Write the sum back to shared memory
        temp[threadIdx.x] += val;
        __syncthreads(); // Synchronize to ensure all writes are complete
    }

    // Write the result to global memory
    out[gid] = temp[threadIdx.x];
}