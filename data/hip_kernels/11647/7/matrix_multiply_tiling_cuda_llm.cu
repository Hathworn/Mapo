#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_multiply_tiling_cuda(int* A, int* B, int* C, int m, int n) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = n * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd = aBegin + n - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep = BLOCK_SIZE * m;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    int Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {

        // Declaration of the shared memory array As used to
        // store the sub-matrix of A
        __shared__ int As[BLOCK_SIZE][BLOCK_SIZE];

        // Declaration of the shared memory array Bs used to
        // store the sub-matrix of B
        __shared__ int Bs[BLOCK_SIZE][BLOCK_SIZE];

        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        int aIndex = a + n * ty + tx;
        int bIndex = b + m * ty + tx;
        if (aIndex < m * n) As[ty][tx] = A[aIndex];  // Avoid out-of-bounds access
        if (bIndex < m * n) Bs[ty][tx] = B[bIndex];  // Avoid out-of-bounds access

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Csub += As[ty][k] * Bs[k][tx];
        }

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = m * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    if (c + m * ty + tx < m * m) C[c + m * ty + tx] = Csub;  // Avoid out-of-bounds access
}