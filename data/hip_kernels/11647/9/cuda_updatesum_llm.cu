#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_updatesum(int *array, int *update_array, int size)
{
    extern __shared__ int shared[];

    unsigned int tid = threadIdx.x;
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load previous block sum into a register for faster access
    int op = (blockIdx.x > 0) ? update_array[blockIdx.x - 1] : 0;

    // Avoid out-of-bounds access
    if (id < size) {
        shared[tid] = array[id] + op;
        __syncthreads(); // Ensure all threads have written to shared memory
        array[id] = shared[tid];
    }
}