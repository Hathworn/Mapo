#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int* in, int* out, int offset, int n) {
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    // Use conditional operator for concise increment logic
    if (gid < n) {
        out[gid] = in[gid] + ((gid >= offset) ? in[gid - offset] : 0);
    }
}