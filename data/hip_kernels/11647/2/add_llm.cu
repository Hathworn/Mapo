#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int* in, int* out, int offset, int n) {
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    // Use branch-less assignment with conditional operator
    out[gid] = (gid < n) ? in[gid] + ((gid >= offset) ? in[gid - offset] : 0) : 0;
}