#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(double* in, double* out, int offset, int n) {
    int gid = threadIdx.x + blockIdx.x * blockDim.x;

    // Exit early if the thread is out of bounds
    if (gid >= n) return; 

    // Perform the addition in a conditional operation
    double temp = in[gid];
    if (gid >= offset) {
        temp += in[gid-offset];
    }

    // Coalesced memory write
    out[gid] = temp;
}