#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int* in, int offset, int n) {
    int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= n) return;
    
    extern __shared__ int temp[];
    
    // Load data into shared memory
    temp[threadIdx.x] = in[gid];
    
    __syncthreads(); // Ensure all data is loaded before proceeding
    
    if (gid >= offset) {
        // Use shared memory to reduce global memory accesses
        in[gid] += temp[threadIdx.x - (gid >= blockDim.x ? offset : 0)];
    }
    
    // Store result back into global memory
    in[gid] = temp[threadIdx.x];
}