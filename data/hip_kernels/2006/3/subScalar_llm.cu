#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use extern "C" only once as it's redundant to declare it multiple times.
extern "C"

/*
* Perform a subtraction of scalar 'b' from each element in array 'a' of length 'n',
* Results are stored in 'result'.
*/
__global__ void subScalar(int n, float *a, float b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (i < n)
    {
        // Perform scalar subtraction
        result[i] = a[i] - b;
    }
}