#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to perform element-wise division of two arrays
extern "C"
__global__ void cuDiv(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // Perform division only if the divisor is not zero
        float div = b[i];
        result[i] = div != 0.0f ? a[i] / div : 0.0f; // Avoid division by zero
    }
}