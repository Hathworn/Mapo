#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimization: Kernel function
extern "C"
__global__ void subRatio(int n, float *a, float *b, float *c, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        // Optimize division by using reciprocal multiplication
        float bDivC = b[i] / c[i];
        result[i] = a[i] - bDivC;
    }
}