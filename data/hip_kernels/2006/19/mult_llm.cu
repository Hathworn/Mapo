#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Perform a reduction from data of length 'size' to result, where length of result will be 'number of blocks'.
*/
extern "C"
__global__ void mult(int n, float *a, float *b, float *result)
{
    // Use shared memory for partial results
    extern __shared__ float shared[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (i < n)
        shared[threadIdx.x] = a[i] * b[i];
    else
        shared[threadIdx.x] = 0.0f; // Handle out-of-bounds

    __syncthreads(); // Ensure all threads have loaded data

    // Reduction within block
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride)
            shared[threadIdx.x] += shared[threadIdx.x + stride];
        __syncthreads(); // Sync after each reduction step
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0)
        result[blockIdx.x] = shared[0];
}