#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access by using shared memory for reduction
extern "C"
__global__ void accrue(int n, float *a, float *b, float p, float *result)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for performance improvement
    extern __shared__ float sharedData[];
    
    // Perform calculation if within bounds
    if (i < n)
    {
        sharedData[threadIdx.x] = a[i] * (1.0f + b[i] * p);
    }
    else
    {
        sharedData[threadIdx.x] = 0.0f; // Ensure all threads write to shared memory
    }
    __syncthreads();
    
    // Ensure all shared memory writes are completed
    if (i < n) 
    {
        result[i] = sharedData[threadIdx.x];
    }
}