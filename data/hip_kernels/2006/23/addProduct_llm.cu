#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
extern "C"
__global__ void addProduct(int n, float *a, float *b, float *c, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process the elements within bounds
    if (i < n)
    {
        result[i] = a[i] + b[i] * c[i];
    }
    
    // Utilize all available threads by processing additional elements if needed
    for (int j = i + blockDim.x * gridDim.x; j < n; j += blockDim.x * gridDim.x)
    {
        result[j] = a[j] + b[j] * c[j];
    }
}