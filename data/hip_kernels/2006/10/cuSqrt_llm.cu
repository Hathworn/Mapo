#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to compute square roots of array elements
extern "C"
__global__ void cuSqrt(int n, float *a, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // Use fast math for optimization
        result[i] = __fsqrt_rn(a[i]);
    }
}