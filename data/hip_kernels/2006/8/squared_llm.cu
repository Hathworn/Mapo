#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function for squaring elements
extern "C"
__global__ void squared(int n, float *a, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for optimization
    for (; i < n; i += stride) { // Loop with stride to improve data access pattern
        result[i] = a[i] * a[i];
    }
}