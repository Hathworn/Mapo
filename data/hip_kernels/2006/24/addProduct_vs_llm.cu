#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Perform a reduction from data of length 'size' to result, where length of result will be 'number of blocks'.
*/
extern "C"
__global__ void addProduct_vs(int n, float *a, float *b, float c, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    int gridSize = blockDim.x * gridDim.x;          // Calculate total number of threads in grid

    // Loop through all elements, processing those assigned to current thread index
    while (i < n) {
        result[i] = a[i] + b[i] * c;
        i += gridSize;                              // Move to next element in grid stride
    }
}