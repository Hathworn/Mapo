#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

/*
* Perform a reduction from data of length 'n' to result, where length of result will be 'number of blocks'.
*/
__global__ void cap(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Process only when index is within valid range
    if (i < n)
    {
        // Use ternary operator to select the minimum value
        result[i] = fminf(a[i], b[i]); // Use fminf for potentially better performance
    }
}