#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for vector subtraction
extern "C"
__global__ void sub(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop to handle larger arrays, increasing parallelism
    for (int idx = i; idx < n; idx += gridDim.x * blockDim.x)
    {
        result[idx] = a[idx] - b[idx];
    }
}