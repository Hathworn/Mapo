#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void floorByScalar(int n, float *a, float b, float *result)
{
    // Cache calculations to reduce repeated operations
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use warp-synchronous instruction if block size is a multiple of warp size
    if (i < n)
    {
        float ai = a[i];        // Load once
        result[i] = ai > b ? ai : b;   // Use conditional to store result
    }
}