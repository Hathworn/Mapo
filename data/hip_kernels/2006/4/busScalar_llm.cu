#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" __global__ void busScalar(int n, float *a, float b, float *result)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform operation if index is within bounds
    if (i < n)
    {
        result[i] = b - a[i]; // Simplify operation order
    }
}