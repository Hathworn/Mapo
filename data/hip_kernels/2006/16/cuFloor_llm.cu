#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
extern "C"
__global__ void cuFloor(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Avoid out-of-bounds access
    if (i < n)
    {
        result[i] = fmaxf(a[i], b[i]); // Use fmaxf for better performance on floats
    }
}