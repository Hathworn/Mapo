#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
extern "C"
__global__ void addScalar(int n, float *a, float b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure no unnecessary branching
    if (i < n) 
    {
        result[i] = a[i] + b;
    }
}