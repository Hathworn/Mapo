#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

/*
* Perform a reduction from data of length 'size' to result, 
* where length of result will be 'number of blocks'.
*/
extern "C"
__global__ void cuAbs(int n, float *a, float *result)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index is within the bounds
    if (i < n)
    {
        // Use fabsf for floating-point absolute value
        result[i] = fabsf(a[i]);
    }
}