#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Perform a reduction from data of length 'size' to result, where length of result will be 'number of blocks'.
*/
extern "C"
__global__ void vidScalar(int n, float *a, float b, float *result)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we don't access out-of-bounds
    if (i < n)
    {
        result[i] = b / a[i];
    }
}