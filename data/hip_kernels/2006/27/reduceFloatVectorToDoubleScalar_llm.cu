#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Perform a reduction from data of length 'size' to result, where length of result will be 'number of blocks'.
*/
extern "C"
__global__ void reduceFloatVectorToDoubleScalar(int size, void *data, double *result) {
    float *fdata = (float*) data;

    extern __shared__ double sdata[];
    double* s2data = sdata + blockDim.x;
    double* cdata = s2data + blockDim.x;

    // Perform first level of reduction, reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    // Use conditional operator and math functions to minimize divergence and calculate shared data
    double sum1 = (double)((i < size) ? fdata[i] : 0);
    double sum2 = (double)((i + blockDim.x < size) ? fdata[i + blockDim.x] : 0);

    sdata[tid] = sum1 + sum2;
    cdata[tid] = sdata[tid] - sum1 - sum2;
    __syncthreads();

    // Do reduction in shared mem with loop unrolling
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            s2data[tid] = sdata[tid] + sdata[tid + s] - cdata[tid] - cdata[tid + s];
            cdata[tid] = (s2data[tid] - sdata[tid]) - sdata[tid + s];
            sdata[tid] = s2data[tid];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) result[blockIdx.x] = sdata[0];
}