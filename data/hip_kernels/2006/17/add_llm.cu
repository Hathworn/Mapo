#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to perform element-wise addition of two vectors a and b, storing the result in result vector
extern "C"
__global__ void add(int n, float *a, float *b, float *result)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use conditional branch elimination technique
    if (i < n)
    {
        // Perform addition if within bounds
        result[i] = a[i] + b[i];
    }
}