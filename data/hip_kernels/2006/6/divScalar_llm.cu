#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory to avoid redundant memory accesses
extern "C"
__global__ void divScalar(int n, float *a, float b, float *result)
{
    // Compute the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread is within bounds
    if (i < n)
    {
        // Perform division
        result[i] = a[i] / b;
    }
}