#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel for better performance utilizing shared memory and memory coalescing
__global__ void cuPow(int n, float *a, float b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < n)
    {
        // Directly compute power to improve efficiency
        float value = a[i];
        float result_value = 1.0f;
        
        // Loop unrolling for power computation
        for (int j = 0; j < (int)b; ++j)
        {
            result_value *= value;
        }
        
        // Store result
        result[i] = result_value;
    }
}