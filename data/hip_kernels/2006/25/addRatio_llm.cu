#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void addRatio(int n, float *a, float *b, float *c, float *result)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use conditional to ensure accessing valid memory
    if (i < n)
    {
        // Store the result of element-wise computation
        result[i] = a[i] + b[i] / c[i];
    }
}