#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function for inverting array elements.
extern "C"
__global__ void invert(int n, float *a, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;  // Calculate stride for each thread
    for (int idx = i; idx < n; idx += stride) {  // Loop over elements with stride
        result[idx] = 1.0f / a[idx];
    }
}