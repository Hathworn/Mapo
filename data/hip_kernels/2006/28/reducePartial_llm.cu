#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void reducePartial(int size, void *data, void *result) {
    float *fdata = (float*) data;
    float *sum = (float*) result;

    extern __shared__ double sdata[];
    // Utilize fewer shared memory allocations
    double* cdata = sdata + 2 * blockDim.x;

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    
    // Load data to shared memory with bounds check reduction
    double temp1 = (i < size) ? fdata[i] : 0.0;
    double temp2 = (i + blockDim.x < size) ? fdata[i + blockDim.x] : 0.0;
    sdata[tid] = temp1 + temp2;
    cdata[tid] = sdata[tid] - temp1 - temp2;
    __syncthreads();

    // Perform parallel reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            double pe = sdata[tid];
            double qe = sdata[tid + s];
            double cs = cdata[tid];
            sdata[tid] = pe + qe - cs - cdata[tid + s];
            cdata[tid] = (sdata[tid] - pe) - qe;
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) sum[blockIdx.x] = sdata[0];
}