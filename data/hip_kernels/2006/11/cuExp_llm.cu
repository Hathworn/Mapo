#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to compute exponentials
extern "C"
__global__ void cuExp(int n, float *a, float *result)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if index is within bounds
    if (i < n)
    {
        // Compute exponential and store the result
        result[i] = expf(a[i]); // Use expf for better performance on float
    }
}