#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Perfom a reduction from data of length 'size' to result, where length of result will be 'number of blocks'. */
extern "C"
__global__ void multScalar(int n, float *a, float b, float *result)
{
    // Use grid-stride loop for better memory access and load balancing
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        result[i] = a[i] * b;
    }
}