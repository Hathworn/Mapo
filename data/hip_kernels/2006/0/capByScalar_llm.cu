#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for capping values by a scalar
extern "C"
__global__ void capByScalar(int n, float *a, float b, float *result)
{
    // Load cap into a register for faster access
    float cap = b;
    
    // Calculate global index for current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread does not exceed array bounds
    if (i < n)
    {
        // Perform capping in a single step
        result[i] = fminf(a[i], cap);
    }
}