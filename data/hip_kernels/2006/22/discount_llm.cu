#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Perfom a reduction from data of length 'size' to result, where length of result will be 'number of blocks'.
*/
extern "C"
__global__ void discount(int n, float *a, float *b, float p, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure that 'i' is within bounds
    if (i < n) 
    {
        // Combine multiply and addition operation in a single line to optimize performance
        float prod = b[i] * p;
        float fma = __fmaf_rn(prod, 1.0f, 1.0f);
        result[i] = a[i] / fma;
    }
}