#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for logarithm computation
extern "C"
__global__ void cuLog(int n, float *a, float *result)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure work is within bounds and perform operation
    if (i < n)
    {
        result[i] = logf(a[i]); // Use logf for float calculations
    }
}