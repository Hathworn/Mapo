#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 128

__global__ void calc_freq(int *freq, int file_size, char *buffer, int total_threads){
    // Allocate shared memory for block-level frequency accumulation
    __shared__ int shared_freq[N];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int local_idx = threadIdx.x;
    
    // Initialize shared memory
    if (local_idx < N) {
        shared_freq[local_idx] = 0;
    }
    __syncthreads();
    
    // Calculate per-thread local frequency
    for (int i = idx; i < file_size; i += total_threads) {
        atomicAdd(&shared_freq[buffer[i]], 1);
    }
    __syncthreads();
    
    // Reduce from shared memory to global memory
    if (local_idx < N) {
        atomicAdd(&freq[local_idx], shared_freq[local_idx]);
    }
}