#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calc(float *result, float *b, float *a, int size) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        float temp = 0.0f;

        // Use shared memory for 'b' to reduce global memory accesses
        __shared__ float shared_b[1024]; // Assume max blockDim.x is 1024

        // Load data from 'b' into shared memory
        if (threadIdx.x < size) {
            shared_b[threadIdx.x] = b[threadIdx.x];
        }
        __syncthreads();

        // Accumulate results using shared memory
        for (int j = 0; j < size; j++) {
            temp += a[j + (idx * size)] * shared_b[j];
        }
        
        // Atomic add the result
        atomicAdd(&result[idx], temp);
    }
}