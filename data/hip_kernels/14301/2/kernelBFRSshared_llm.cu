#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float maxMetricPoints(const float* g_uquery, const float* g_vpoint, int pointdim, int signallength) {
    float r_u1, r_v1, r_d1, r_dim = 0;  // local variables

    for(int d = 0; d < pointdim; d++) {
        r_u1 = g_uquery[d * signallength];
        r_v1 = g_vpoint[d * signallength];
        r_d1 = fabsf(r_v1 - r_u1);  // use fabsf for absolute value
        
        r_dim = fmaxf(r_dim, r_d1);  // use fmaxf for max comparison
    }
    return r_dim;
}

__global__ void kernelBFRSshared(const float* g_uquery, const float* g_vpointset, int *g_npoints, int pointdim, int triallength, int signallength, int exclude, float radius) {
    extern __shared__ char array[];
    int *s_npointsrange = (int*)array;
    
    const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned int itrial = tid / triallength;  // indextrial
    
    if(tid < signallength) {
        s_npointsrange[threadIdx.x] = 0;
        __syncthreads();
        
        unsigned int indexi = tid - triallength * itrial;
        
        for(int t = 0; t < triallength; t++) {
            int indexu = tid;
            int indexv = t + itrial * triallength;
            int condition1 = indexi - exclude;
            int condition2 = indexi + exclude;
            
            if((t < condition1) || (t > condition2)) {
                float temp_dist = maxMetricPoints(g_uquery + indexu, g_vpointset + indexv, pointdim, signallength);
                
                if(temp_dist <= radius) {
                    atomicAdd(&s_npointsrange[threadIdx.x], 1);  // atomic increment to ensure correctness
                }
            }
        }
        
        __syncthreads();
        
        // Copy to global memory
        g_npoints[tid] = s_npointsrange[threadIdx.x];
    }
}