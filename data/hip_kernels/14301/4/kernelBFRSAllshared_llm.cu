#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float maxMetricPoints(const float* g_uquery, const float* g_vpoint, int pointdim, int signallength) {
    float r_dim = 0;
    for (int d = 0; d < pointdim; d++) {
        float r_u1 = g_uquery[d * signallength];
        float r_v1 = g_vpoint[d * signallength];
        float r_d1 = fabsf(r_v1 - r_u1);  // Use intrinsic for abs
        r_dim = fmaxf(r_dim, r_d1);
    }
    return r_dim;
}

__global__ void kernelBFRSAllshared(const float* g_uquery, const float* g_vpointset, int* g_npoints, int pointdim, int triallength, int signallength, int exclude, const float* vecradius) {

    // Shared memory
    extern __shared__ int s_npointsrange[];
    float radius = 0;
    const unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    const unsigned int itrial = tid / triallength;  // index trial

    if (tid < signallength) {

        s_npointsrange[threadIdx.x] = 0;
        __syncthreads();

        radius = vecradius[tid];
        unsigned int indexi = tid - triallength * itrial;

        for (int t = 0; t < triallength; t++) {
            int indexu = tid;
            int indexv = t + itrial * triallength;
            int condition1 = indexi - exclude;
            int condition2 = indexi + exclude;

            if ((t < condition1) || (t > condition2)) {
                float temp_dist = maxMetricPoints(g_uquery + indexu, g_vpointset + indexv, pointdim, signallength);
                if (temp_dist <= radius) {
                    s_npointsrange[threadIdx.x]++;
                }
            }
        }

        __syncthreads();
        // Copy to global memory
        g_npoints[tid] = s_npointsrange[threadIdx.x];
    }
}