#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float maxMetricPoints(const float* g_uquery, const float* g_vpoint, int pointdim, int signallength) {
    float r_u1, r_v1, r_d1, r_dim = 0;

    #pragma unroll  // Optimize loop unrolling
    for (int d = 0; d < pointdim; d++) {
        r_u1 = g_uquery[d * signallength];
        r_v1 = g_vpoint[d * signallength];
        r_d1 = fabsf(r_v1 - r_u1);  // Use optimized absolute value function
        r_dim = fmaxf(r_dim, r_d1);  // Use optimized max function
    }
    return r_dim;
}

__global__ void kernelBFRSMultishared(const float* g_uquery, const float* g_vpointset, int *g_npoints, int pointdim, int triallength, int signallength, int exclude, const float* vecradius) {

    extern __shared__ char array[];
    int *s_npointsrange = (int*)array;
    float radius = 0;
    const unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    const unsigned int itrial = tid / triallength;

    if (tid < signallength) {
        s_npointsrange[threadIdx.x] = 0;
        __syncthreads();

        radius = vecradius[itrial];
        unsigned int indexi = tid - triallength * itrial;

        for (int t = 0; t < triallength; t++) {
            int indexv = t + itrial * triallength;
            int condition1 = indexi - exclude;
            int condition2 = indexi + exclude;

            if (t < condition1 || t > condition2) {
                float temp_dist = maxMetricPoints(g_uquery + tid, g_vpointset + indexv, pointdim, signallength);
                if (temp_dist <= radius) {
                    s_npointsrange[threadIdx.x]++;
                }
            }
        }

        __syncthreads();
        g_npoints[tid] = s_npointsrange[threadIdx.x];
    }
}