#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float maxMetricPoints(const float* g_uquery, const float* g_vpoint, int pointdim, int signallength) {
    float r_u1, r_v1, r_d1, r_dim = 0;

    // Use pointer arithmetic instead of array indexing
    for (int d = 0; d < pointdim; d++) {
        r_u1 = g_uquery[d * signallength];
        r_v1 = g_vpoint[d * signallength];
        r_d1 = fabsf(r_v1 - r_u1);  // Use fabsf for absolute value
        r_dim = fmaxf(r_dim, r_d1); // Use fmaxf for maximum
    }
    return r_dim;
}

__device__ float insertPointKlist(int kth, float distance, int indexv, float* kdistances, int* kindexes) {
    int k = 0;
    // Simplify while loop using internal logic
    while (k < kth-1 && distance > kdistances[k]) { 
        k++;
    }
    // Use memmove instead of loop for shifting
    if (k < kth) {
        memmove(kdistances + k + 1, kdistances + k, (kth - k - 1) * sizeof(float));
        memmove(kindexes + k + 1, kindexes + k, (kth - k - 1) * sizeof(int));
        // Replace
        kdistances[k] = distance;
        kindexes[k] = indexv;
    }
    return kdistances[kth - 1];
}

__global__ void kernelKNNshared(const float* g_uquery, const float* g_vpointset, int *g_indexes, float* g_distances, const int pointdim, const int triallength, const int signallength, const int kth, const int exclude) {
    extern __shared__ char array[];
    float *kdistances = (float*)array;
    int *kindexes = (int*)(array + kth * blockDim.x * sizeof(float));

    const unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    const unsigned int itrial = tid / triallength;

    if (tid < signallength) {
        // Initialize shared memory with INFINITY
        for (int k = 0; k < kth; k++) {
            kdistances[threadIdx.x * kth + k] = INFINITY;
            kindexes[threadIdx.x * kth + k] = -1; // Initialize kindexes
        }

        __syncthreads();

        float r_kdist = INFINITY;
        unsigned int indexi = tid - triallength * itrial;

        for (int t = 0; t < triallength; t++) {
            int indexu = tid;
            int indexv = (t + itrial * triallength);
            int condition1 = indexi - exclude;
            int condition2 = indexi + exclude;
            
            // Reduce branching by calculating both conditions in one go
            if ((t < condition1) || (t > condition2)) {
                float temp_dist = maxMetricPoints(g_uquery + indexu, g_vpointset + indexv, pointdim, signallength);
                if (temp_dist <= r_kdist) {
                    r_kdist = insertPointKlist(kth, temp_dist, t, kdistances + threadIdx.x * kth, kindexes + threadIdx.x * kth);
                }
            }
        }
        
        __syncthreads();

        // COPY TO GLOBAL MEMORY
        for (int k = 0; k < kth; k++) {
            g_indexes[tid + k * signallength] = kindexes[threadIdx.x * kth + k];
            g_distances[tid + k * signallength] = kdistances[threadIdx.x * kth + k];
        }
    }
}