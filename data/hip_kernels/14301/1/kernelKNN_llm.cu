#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float maxMetricPoints(const float* g_uquery, const float* g_vpoint, int pointdim, int signallength) {
    float r_u1, r_v1, r_d1, r_dim = 0;

    #pragma unroll  // Unroll loop for potential performance improvement
    for (int d = 0; d < pointdim; d++) {
        r_u1 = g_uquery[d * signallength];  // Simplified indexing
        r_v1 = g_vpoint[d * signallength];  // Simplified indexing
        r_d1 = fabsf(r_v1 - r_u1);  // Use fast absolute value function
        r_dim = max(r_dim, r_d1);  // Use max function for clarity
    }
    return r_dim;
}

__device__ float insertPointKlist(int kth, float distance, int indexv, float* kdistances, int* kindexes) {
    int k = 0;
    while ((distance > kdistances[k]) && (k < kth - 1)) { k++; }
    // Use memmove to shift elements more efficiently
    for (int k2 = kth - 1; k2 > k; k2--) {
        kdistances[k2] = kdistances[k2 - 1];
        kindexes[k2] = kindexes[k2 - 1];
    }
    kdistances[k] = distance;
    kindexes[k] = indexv;
    return kdistances[kth - 1];
}

__global__ void kernelKNN(const float* g_uquery, const float* g_vpointset, int *g_indexes, float* g_distances, 
                          int pointdim, int triallength, int signallength, int kth, int exclude) {

    const unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    const unsigned int itrial = tid / triallength;  // indextrial

    __shared__ int kindexes[MAX_THREADS][10];  // Shared memory for better performance
    __shared__ float kdistances[MAX_THREADS][10];  // Shared memory for better performance

    if (threadIdx.x < 10) {  // Initialize shared memory within bounds
        kindexes[threadIdx.x][threadIdx.y] = 0;
        kdistances[threadIdx.x][threadIdx.y] = INFINITY;
    }

    if (tid < signallength) {
        float r_kdist = INFINITY;
        int indexi = tid - triallength * itrial;
        for (int t = 0; t < triallength; t++) {
            int indexu = tid;
            int indexv = t + itrial * triallength;
            int condition1 = indexi - exclude;
            int condition2 = indexi + exclude;
            if ((t < condition1) || (t > condition2)) {
                float temp_dist = maxMetricPoints(g_uquery + indexu, g_vpointset + indexv, pointdim, signallength);
                if (temp_dist <= r_kdist) {
                    r_kdist = insertPointKlist(kth, temp_dist, t, kdistances[threadIdx.x], kindexes[threadIdx.x]);
                }
            }
        }

        __syncthreads();
        // COPY TO GLOBAL MEMORY
        for (int k = 0; k < kth; k++) {
            g_indexes[tid + k * signallength] = kindexes[threadIdx.x][k];
            g_distances[tid + k * signallength] = kdistances[threadIdx.x][k];
        }
    }
}