#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDivideByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate number of threads working in parallel
    const unsigned int numThreads = blockDim.x * gridDim.x;
    // Loop through data in strides of numThreads
    for (unsigned int i = idx; i < width * height; i += numThreads) {
        // Perform element-wise division, take vec's corresponding column value using i / width
        tgtMat[i] = mat[i] / vec[i / width];
    }
}