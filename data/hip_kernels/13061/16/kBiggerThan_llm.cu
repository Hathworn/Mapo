#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBiggerThan(float* gMat1, float* gMat2, float* gMatTarget, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Evaluate condition and assign result within bounds check
    if (idx < numElements) {
        gMatTarget[idx] = (gMat1[idx] > gMat2[idx]) ? 1.0f : 0.0f;
    }
}