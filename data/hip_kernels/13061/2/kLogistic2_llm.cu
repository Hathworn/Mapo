#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogistic2(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Cache grid and block dimension to minimize register usage inside loop
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numElements; i += stride) {
        // Reduce function calls by storing result in a variable
        float exp_val = expf(-gData[i]);
        target[i] = 1 / (1 + exp_val);
    }
}