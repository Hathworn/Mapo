#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kZero(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a simple stride loop for efficient memory access
    if (idx < numElements) {
        for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x) {
            target[i] = 0;
        }
    }
}