#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTranspose(float* a, float* dest, int width, int height) {
    const int bx = blockIdx.x * blockDim.x;
    const int by = blockIdx.y * blockDim.y;
    const int tx = bx + threadIdx.x;
    const int ty = by + threadIdx.y;

    __shared__ float smem[ADD_BLOCK_SIZE][ADD_BLOCK_SIZE + 1];

    // Load matrix data into shared memory for coalesced reads
    if (tx < width && ty < height) {
        smem[threadIdx.y][threadIdx.x] = a[ty * width + tx];
    }
    __syncthreads();

    // Store transposed data back into global memory
    if ((by + threadIdx.x) < height && (bx + threadIdx.y) < width) {
        dest[(bx + threadIdx.y) * height + by + threadIdx.x] = smem[threadIdx.x][threadIdx.y];
    }
}