#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSquaredDiff(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Efficiently iterate over elements by stride based on all threads
    if (idx < numEls) {
        dest[idx] = (a[idx] - b[idx]) * (a[idx] - b[idx]);
    }
}