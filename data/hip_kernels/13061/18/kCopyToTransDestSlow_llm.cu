#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline int getTransArrayIndex(unsigned int width, unsigned int height, unsigned int i) {
    return height * (i % width) + i / width;
}

__global__ void kCopyToTransDestSlow(float* srcStart, float* destStart, unsigned int srcCopyWidth, unsigned int srcJumpWidth, unsigned int destJumpHeight, unsigned int numElements) {
    // Compute unique thread index consistently across blocks
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numElements) {
        // Calculate row and column for optimized memory access
        unsigned int srcRow = idx / srcCopyWidth;
        unsigned int srcCol = idx % srcCopyWidth;
        unsigned int transIndex = destJumpHeight * srcCol + srcRow;

        // Use transIndex to access transposed memory efficiently
        destStart[transIndex] = srcStart[srcRow * srcJumpWidth + srcCol];
    }
}