#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kMultByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate the linear index for the current thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by using a proper grid-stride loop for better occupancy
    unsigned int stride = blockDim.x * gridDim.x;

    // Iterate over the matrix with grid-stride loop
    while (idx < width * height) {
        tgtMat[idx] = mat[idx] * vec[idx % width];
        idx += stride;
    }
}