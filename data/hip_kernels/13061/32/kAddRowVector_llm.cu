#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height, float scaleVec) {
    // Calculate the global index based directly on block and thread indices
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds (optimization: remove redundant calculations)
    if (idx < width * height) {
        // Use shared memory if possible (optional further optimization depending on data access patterns)
        unsigned int row = idx / width;

        // Compute and update the target matrix value
        tgtMat[idx] = mat[idx] + scaleVec * vec[idx % width];
    }
}
```
