#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kBiggerThanScalar(float* gData, float scalar, float* target, unsigned int numElements) {
    // Calculate global index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for improved parallel execution
    for (; idx < numElements; idx += blockDim.x * gridDim.x) {
        target[idx] = gData[idx] > scalar;
    }
}