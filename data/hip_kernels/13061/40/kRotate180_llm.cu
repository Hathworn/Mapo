#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kRotate180(float* filters, float* targets, const int filterSize) {
    const int filtIdx = blockIdx.x;
    const int readStart = filterSize * filterSize * filtIdx;
    filters += readStart;
    targets += readStart;

    // Optimize: Use single indexing instead of nested loops.
    for(int idx = threadIdx.y * filterSize + threadIdx.x; idx < filterSize * filterSize; idx += 256) {
        const int x = idx % filterSize;
        const int y = idx / filterSize;
        const int writeX = filterSize - 1 - x;
        const int writeY = filterSize - 1 - y;

        targets[writeY * filterSize + writeX] = filters[y * filterSize + x];
    }
}