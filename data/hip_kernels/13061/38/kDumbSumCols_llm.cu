#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDumbSumCols(float* mat, float* vec, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < width) {
        float sum = 0;
        for (int j = 0; j < height; j++) {
            // Access mat using optimized pointer arithmetic to reduce pointer calculations
            sum += mat[j * width + idx];
        }
        vec[idx] = sum;
    }
}