#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCopyToTransDestFast(float* srcStart, float* destStart, unsigned int srcCopyWidth, unsigned int srcCopyHeight, unsigned int srcJumpSize, unsigned int destJumpSize) {
    const unsigned int idxY = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int idxX = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit for threads outside the bounds.
    if (idxX >= srcCopyWidth || idxY >= srcCopyHeight) return;

    const unsigned int srcReadIdx = idxY * srcJumpSize + idxX;
    const unsigned int destWriteIdx = (blockIdx.x * blockDim.x + threadIdx.y) * destJumpSize + blockIdx.y * blockDim.y + threadIdx.x;
    __shared__ float smem[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE + 1];

    // Load input data into shared memory
    smem[threadIdx.y][threadIdx.x] = srcStart[srcReadIdx];
    __syncthreads();

    // Transpose and write data to destination
    destStart[destWriteIdx] = smem[threadIdx.x][threadIdx.y];
}