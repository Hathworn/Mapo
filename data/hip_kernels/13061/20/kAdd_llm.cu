#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void kAdd(float* a, float* b, float* dest, unsigned int numEls, float scaleA, float scaleB) {
    // Use shared memory to reduce global memory accesses
    __shared__ float sharedA[256];
    __shared__ float sharedB[256];

    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        // Load data into shared memory
        sharedA[threadIdx.x] = a[i];
        sharedB[threadIdx.x] = b[i];

        __syncthreads();

        // Perform the computation using shared memory
        dest[i] = scaleA * sharedA[threadIdx.x] + scaleB * sharedB[threadIdx.x];

        __syncthreads();
    }
}