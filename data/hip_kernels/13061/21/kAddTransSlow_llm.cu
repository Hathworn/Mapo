#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline int getTransArrayIndex(unsigned int width, unsigned int height, unsigned int i) {
    return height * (i % width) + i / width;
}

__global__ void kAddTransOptimized(float* a, float* b, float* dest, unsigned int width, unsigned int height, unsigned int numEls, float scaleA, float scaleB) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Optimize memory coalescing by accessing data sequentially
    for (unsigned int i = idx; i < numEls; i += numThreads) {
        unsigned int transIndex = getTransArrayIndex(width, height, i);
        float bValue = (transIndex < numEls) ? b[transIndex] : 0.0f; // Prevent out-of-bounds in case
        dest[i] = scaleA * a[i] + scaleB * bValue;
    }
}