#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSquare(float* gData, float* target, unsigned int numElements) {
    // Calculate initial global index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stepSize = blockDim.x * gridDim.x; // Calculate step size only once

    for (unsigned int i = idx; i < numElements; i += stepSize) {
        target[i] = gData[i] * gData[i]; // Perform element-wise squaring
    }
}