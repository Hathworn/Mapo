#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void kMultTransFast(float* a, float* b, float* dest, unsigned int width, unsigned int height, unsigned int bJumpWidth) {
    const unsigned int idxY = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int idxX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idx = idxY * width + idxX;

    // Use bank conflict-free shared memory access
    __shared__ float smem[ADD_BLOCK_SIZE][ADD_BLOCK_SIZE + 1];

    // Calculate the starting position for reading from 'b'
    const unsigned int bBlockReadStart = blockDim.x * blockIdx.x * bJumpWidth + blockIdx.y * blockDim.y;

    // Load data into shared memory
    if (idxX < width && idxY < height) {
        smem[threadIdx.y][threadIdx.x] = b[bBlockReadStart + threadIdx.y * bJumpWidth + threadIdx.x];
    }
    __syncthreads();

    // Perform the element-wise multiplication
    if (idxX < width && idxY < height) {
        dest[idx] = a[idx] * smem[threadIdx.y][threadIdx.x];
    }
}