#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(float* src, float* tgt, unsigned int srcWidth, unsigned int srcHeight, unsigned int tgtWidth, unsigned int tgtHeight) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Use faster integer arithmetic for index calculations
    for (unsigned int i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const unsigned int srcY = (i / tgtWidth) % srcHeight;
        const unsigned int srcX = (i % tgtWidth) % srcWidth;
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}