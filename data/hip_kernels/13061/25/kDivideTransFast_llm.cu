#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivideTransFast(float* a, float* b, float* dest, unsigned int width, unsigned int height, unsigned int bJumpWidth) {
    const unsigned int idxY = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int idxX = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit if thread indices exceed bounds.
    if (idxY >= height || idxX >= width) return;

    const unsigned int idx = idxY * width + idxX;

    __shared__ float smem[ADD_BLOCK_SIZE][ADD_BLOCK_SIZE + 1];

    const unsigned int bBlockReadStart = blockDim.x * blockIdx.x * bJumpWidth + blockIdx.y * blockDim.y;

    // Load b into shared memory and ensure all threads have loaded the data.
    if (threadIdx.y < bJumpWidth && (bBlockReadStart + threadIdx.y * bJumpWidth + threadIdx.x) < height * bJumpWidth) {
        smem[threadIdx.x][threadIdx.y] = b[bBlockReadStart + threadIdx.y * bJumpWidth + threadIdx.x];
    }
    __syncthreads();

    // Perform division, using shared memory.
    dest[idx] = __fdividef(a[idx], smem[threadIdx.y][threadIdx.x]);
}