#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate the row and column indices
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    // Loop over columns in a stride pattern
    for (unsigned int i = col; i < width; i += stride) {
        for (unsigned int j = 0; j < height; ++j) {
            unsigned int idx = j * width + i; // Linear index
            tgtMat[idx] = mat[idx] * vec[j]; // Use row index to access vec
        }
    }
}