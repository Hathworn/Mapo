#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddColVector(float* mat, float* vec, float* tgtMat, const unsigned int width, const unsigned int height, const float scaleVec) {
    const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;  // Calculate row index
    const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate column index

    if (row < height && col < width) {  // Check bounds
        unsigned int idx = row * width + col;  // Calculate linear index
        tgtMat[idx] = mat[idx] + scaleVec * vec[row];  // Perform operation
    }
}
```
