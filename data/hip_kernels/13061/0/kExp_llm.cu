#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kExp(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop for improved memory throughput
    for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x) {
        target[i] = __expf(gData[i]);
    }
}