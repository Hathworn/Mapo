#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSubtractFromScalar(float* gData, float scalar, float* target, unsigned int numElements) {
    // Cache thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Iterate over each element assigned to this thread, process in strides
    for (unsigned int i = idx; i < numElements; i += stride) {
        target[i] = scalar - gData[i];
    }
}