#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd3(float* a, const float* b, const float* c, const unsigned int numEls, const float scaleA, const float scaleB, const float scaleC) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll the loop for improved performance
    if (idx < numEls) {
        a[idx] = scaleA * a[idx] + scaleB * b[idx] + scaleC * c[idx];
        const unsigned int numThreads = blockDim.x * gridDim.x;
        unsigned int nextIdx = idx + numThreads;
        while (nextIdx < numEls) {
            a[nextIdx] = scaleA * a[nextIdx] + scaleB * b[nextIdx] + scaleC * c[nextIdx];
            nextIdx += numThreads;
        }
    }
}