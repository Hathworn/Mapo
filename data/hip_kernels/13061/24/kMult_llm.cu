#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMult(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use grid-stride loop to ensure all elements are processed
    for (unsigned int i = idx; i < numEls; i += blockDim.x * gridDim.x) {
        dest[i] = a[i] * b[i]; // Perform element-wise multiplication
    }
}