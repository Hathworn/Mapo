#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddTransFast(float* a, float* b, float* dest, unsigned int width, unsigned int height, unsigned int bJumpWidth, float scaleA, float scaleB) {
    const unsigned int idxY = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int idxX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idx = idxY * width + idxX;

    __shared__ float smem[ADD_BLOCK_SIZE][ADD_BLOCK_SIZE + 1];

    // Coalesced memory access for reading 'b'
    const unsigned int bBlockReadStart = blockDim.x * blockIdx.x * bJumpWidth + blockIdx.y * blockDim.y;
    const unsigned int bIndex = bBlockReadStart + threadIdx.x * bJumpWidth + threadIdx.y;

    smem[threadIdx.y][threadIdx.x] = b[bIndex];
    __syncthreads();

    // Ensure the kernel does not exceed array bounds
    if (idxX < width && idxY < height) {
        dest[idx] = scaleA * a[idx] + scaleB * smem[threadIdx.y][threadIdx.x];
    }
}