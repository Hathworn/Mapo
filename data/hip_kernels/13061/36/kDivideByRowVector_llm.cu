#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivideByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Use shared memory for vector to reduce global memory access
    extern __shared__ float sharedVec[];
    const unsigned int tx = threadIdx.x;
    const unsigned int bx = blockIdx.x;
    const unsigned int bd = blockDim.x;

    if (tx < width) {
        sharedVec[tx] = vec[tx];
    }

    __syncthreads();

    const unsigned int idx = bx * bd + tx;
    const unsigned int numThreads = bd * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        tgtMat[i] = __fdividef(mat[i], sharedVec[i % width]);
    }
}