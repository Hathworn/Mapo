#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLog(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;  // Pre-calculate stride

    for (unsigned int i = idx; i < numElements; i += stride) {
        target[i] = __logf(gData[i]);  // Use cached stride value
    }
}