#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCopy(float* srcStart, float* destStart, unsigned int copyWidth, unsigned int jumpWidth, unsigned int numElements) {
    // Calculate global index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure global index is within bounds
    if (idx < numElements) {
        // Precompute row and column
        unsigned int row = idx / copyWidth;
        unsigned int col = idx % copyWidth;
        unsigned int srcIndex = row * jumpWidth + col;
        
        // Perform copy
        destStart[srcIndex] = srcStart[srcIndex];
    }
}