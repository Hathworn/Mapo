#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSqrt(float* gData, float* target, unsigned int numElements) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop for better utilization of threads
    for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x) {
        target[i] = sqrtf(gData[i]);
    }
}