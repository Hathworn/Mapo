#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogistic1(float* gData, float* target, unsigned int numElements) {
    // Efficient index calculation using grid-stride loop
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int gridSize = blockDim.x * gridDim.x;

    for (; idx < numElements; idx += gridSize) {
        target[idx] = (1 + tanhf(gData[idx] / 2)) / 2;
    }
}