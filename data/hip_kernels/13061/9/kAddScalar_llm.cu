#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddScalar(float* gData, float scalar, float* target, unsigned int numElements) {
    // Calculate the global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    
    // Use shared memory to improve memory access patterns
    __shared__ float sharedData[256]; // Example size, adjust as needed

    // Load data into shared memory
    if (idx < numElements) {
        sharedData[threadIdx.x] = gData[idx];
    }

    // Ensure all threads have loaded their data
    __syncthreads();

    // Perform computation and store result
    if (idx < numElements) {
        target[idx] = scalar + sharedData[threadIdx.x];
    }
}