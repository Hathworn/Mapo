#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kReciprocal(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numElements) {
        target[idx] = 1.0f / gData[idx];  // Use float division for better performance
    }
}