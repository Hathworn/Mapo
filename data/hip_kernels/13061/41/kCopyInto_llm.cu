#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCopyInto(float* images, float* targets, const int imgSize, const int paddingSize, const int numImages) {
    const int imgIdx = blockIdx.y * gridDim.x + blockIdx.x;
    if (imgIdx < numImages) {
        const int targetSize = imgSize + 2 * paddingSize;
        images += imgIdx * imgSize * imgSize;
        targets += imgIdx * targetSize * targetSize + paddingSize * targetSize + paddingSize;

        // Use shared memory for better memory access pattern
        __shared__ float tile[16][16];
        
        for (int y = threadIdx.y; y < imgSize; y += 16) {
            for (int x = threadIdx.x; x < imgSize; x += 16) {
                // Load image data into shared memory
                tile[threadIdx.y][threadIdx.x] = images[y * imgSize + x];
                __syncthreads(); // Synchronize to ensure all data is loaded
                
                // Copy from shared memory to target
                targets[y * targetSize + x] = tile[threadIdx.y][threadIdx.x];
            }
        }
    }
}