#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy(int *dst, int *input_itemsets, int max_rows, int max_cols, int lb0, int lb1, int ub0, int ub1)
{
    // Calculate global row and column indices
    int r = blockIdx.y * blockDim.y + threadIdx.y + lb0;
    int c = blockIdx.x * blockDim.x + threadIdx.x + lb1;

    // Check if indices are within bounds
    if (r < ub0 && c < ub1)
    {
        // Compute 1D index and copy data
        int idx = r * max_cols + c;
        dst[idx] = input_itemsets[idx];
    }
}