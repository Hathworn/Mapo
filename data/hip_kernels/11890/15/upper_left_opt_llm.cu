#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __host__ int maximum(int a, int b, int c) {
    return max(a, max(b, c)); // Simplified maximum function using max
}

__global__ void upper_left_opt(int *dst, int *input_itemsets, int *reference, int max_rows, int max_cols, int i, int penalty) {
    int r = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int c = blockIdx.x * blockDim.x + threadIdx.x + 1;
    
    if (r >= i + 1 || c >= i + 1) return; // Early exit if outside range

    if (blockIdx.x + blockIdx.y >= blockDim.x - 3 && r == (i - c + 1)) { // Single branch condition
        int base = r * max_cols + c;
        dst[base] = maximum(
            input_itemsets[base - max_cols - 1] + reference[base],
            input_itemsets[base - 1] - penalty,
            input_itemsets[base - max_cols] - penalty
        );
    }
}