#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void marshalling1(int *input_itemsets, int *tmp, int max_rows, int max_cols)
{
    // Calculate global thread indices
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Bounds check
    if (i >= max_rows || j >= max_cols) return;

    // Optimize condition: rewriting ternary operation for clarity
    tmp[i * max_cols + j] = (j <= i) ? input_itemsets[(i - j) * max_cols + j] : 0;
}