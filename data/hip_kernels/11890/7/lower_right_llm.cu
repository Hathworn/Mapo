#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimization: Inline maximum function to reduce function call overhead
__device__ __host__ __forceinline__ int maximum_inline(int a, int b, int c) {
    int k = (a <= b) ? b : a;
    return (k <= c) ? c : k;
}

__global__ void lower_right(int *input_itemsets, int *reference, int max_rows, int max_cols, int i, int penalty) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Return immediately if index is out of bound
    if (idx >= i) return;

    int r = max_rows - 1 - idx;
    int c = max_cols - i + idx;
    int base = r * max_cols + c;

    // Use inline maximum function
    input_itemsets[base] = maximum_inline(
        input_itemsets[base - max_cols - 1] + reference[base],
        input_itemsets[base - 1] - penalty,
        input_itemsets[base - max_cols] - penalty
    );
}