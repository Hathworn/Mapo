#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __host__ int maximum(int a, int b, int c) {
    int k;
    if (a <= b)
        k = b;
    else
        k = a;

    if (k <= c)
        return c;
    else
        return k;
}

__global__ void lower_right(int* input_itemsets, int* reference, int* tmp, int max_rows, int max_cols, int i, int penalty) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = i;

    if (c < (max_cols - (i - max_rows + 1))) {
        // Precompute indices to reduce repetitive calculations
        int idx_r_max_cols = r * max_cols;
        int idx_r_minus1_max_cols = (r - 1) * max_cols;
        int idx_r_minus2_max_cols = (r - 2) * max_cols;
        int ref_index = (max_rows - 1 - c) * max_cols + c + (i - max_rows + 1);

        // Perform the calculation using precomputed indices
        tmp[idx_r_max_cols + c] = maximum(
            tmp[idx_r_minus2_max_cols + c + 1] + reference[ref_index],
            tmp[idx_r_minus1_max_cols + c] - penalty,
            tmp[idx_r_minus1_max_cols + c + 1] - penalty
        );
    }
}