#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __host__ int maximum(int a, int b, int c) {
    int max_val = max(max(a, b), c); // Use built-in max function for clarity
    return max_val;
}

__global__ void lower_right_opt(int *dst, int *input_itemsets, int *reference, int max_rows, int max_cols, int i, int penalty) {
    int r = blockIdx.y * blockDim.y + threadIdx.y + i + 1; // Calculate row index
    int c = blockIdx.x * blockDim.x + threadIdx.x + i + 1; // Calculate column index

    if (r >= max_rows || c >= max_cols || r != (max_cols - c + i)) return; // Combine conditions for early return

    // Compute the maximum value and store
    dst[r * max_cols + c] = maximum(
        input_itemsets[(r - 1) * max_cols + c - 1] + reference[r * max_cols + c],
        input_itemsets[r * max_cols + c - 1] - penalty,
        input_itemsets[(r - 1) * max_cols + c] - penalty
    );
}