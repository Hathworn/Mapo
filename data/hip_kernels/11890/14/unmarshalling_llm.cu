#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unmarshalling(int *input_itemsets, int *tmp, int max_rows, int max_cols)
{
    // Calculate global indices
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds
    if (i >= max_rows || j >= max_cols) return;

    // Optimize memory access and computation
    int index = i * max_cols + j;
    int index_in_tmp = (i + j) * max_cols + j;
    if ((i + j) < max_rows) {
        input_itemsets[index] = tmp[index_in_tmp];
    } else {
        input_itemsets[index] = tmp[index_in_tmp - (i + j - max_rows + 1)];
    }
}