#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __host__ int maximum(int a, int b, int c) {
    // Use a concise ternary operation for maximum selection
    return max(max(a, b), c);
}

__global__ void middle(int *input_itemsets, int *reference, int *tmp, int max_rows, int max_cols, int i, int penalty) {
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    // Early return based on condition
    if (c >= (max_cols - 1)) return;

    // Use direct indexing for calculations
    int ref_index = (max_rows - 1 - c) * max_cols + c + 1;
    tmp[max_rows * max_cols + c] = maximum(
        tmp[(max_rows - 2) * max_cols + c] + reference[ref_index],
        tmp[(max_rows - 1) * max_cols + c] - penalty,
        tmp[(max_rows - 1) * max_cols + c + 1] - penalty
    );
}