#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __host__ int maximum(int a, int b, int c) {
    // Simplified maximum calculation
    return max(max(a, b), c);
}

__global__ void upper_left_copy(int *dst, int *input_itemsets, int *reference, int max_rows, int max_cols, int i, int penalty) {
    int r = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int c = blockIdx.x * blockDim.x + threadIdx.x + 1;

    // Early exit if out of bounds
    if (r >= i + 1 || c >= i + 1) return;
    
    int base = r * max_cols + c;
    
    if (r == (i - c + 1)) {
        // Combined reference calculation
        dst[base] = maximum(input_itemsets[base - max_cols - 1] + reference[base],
                           input_itemsets[base - 1] - penalty,
                           input_itemsets[base - max_cols] - penalty);
    } else {
        dst[base] = input_itemsets[base];
    }
}