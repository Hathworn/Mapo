#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __host__ int maximum(int a, int b, int c) {
    // Optimize maximum computation using ternary operator
    return (a >= b) ? ((a >= c) ? a : c) : ((b >= c) ? b : c);
}

__global__ void upper_left(int *input_itemsets, int *reference, int max_rows, int max_cols, int i, int penalty) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= i) return;

    int r = i - idx;
    int c = i + 1 - r;
    int base = r * max_cols + c;

    // Simplified calculation by accessing input_itemsets only once
    int ref_value = reference[base];
    int max_col_c = base - max_cols;
    int input_base = input_itemsets[base];
    int max_base_pos = input_itemsets[max_col_c - 1] + ref_value;

    input_itemsets[base] = maximum(max_base_pos,
                                   input_itemsets[base - 1] - penalty,
                                   input_itemsets[max_col_c] - penalty);
}