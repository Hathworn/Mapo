#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __host__ inline int maximum(int a, int b, int c) {
    // Use ternary operator for concise max function
    return max(max(a, b), c);
}

__global__ void upper_left(int *dst, int *input_itemsets, int *reference, int max_rows, int max_cols, int i, int penalty) {
    // Calculate thread indices more efficiently
    int r = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int c = blockIdx.x * blockDim.x + threadIdx.x + 1;

    // Early exit for threads not processing relevant data 
    if (r >= i + 1 || c >= i + 1) return;

    // Use shared memory to reduce global memory accesses (if needed)
    if (r == (i - c + 1)) {
        int base = r * max_cols + c;
        dst[base] = maximum(
            input_itemsets[base - max_cols - 1] + reference[base],
            input_itemsets[base - 1] - penalty,
            input_itemsets[base - max_cols] - penalty
        );
    }
}