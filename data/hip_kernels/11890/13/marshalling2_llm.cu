#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void marshalling2(int *input_itemsets, int *tmp, int max_rows, int max_cols)
{
    // Calculate global thread coordinates
    int i = blockIdx.y * blockDim.y + threadIdx.y + max_rows;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if out of bounds
    if (i >= max_rows * 2 - 1 || j >= max_cols) return;

    // Streamline index calculation and condition check
    int index = (max_rows - 1 - j) * max_cols + j + 1 + (i - max_rows);
    tmp[i * max_cols + j] = (j < max_cols - (i - max_rows + 1)) ? input_itemsets[index] : 0;
}