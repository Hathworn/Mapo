#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __host__ int maximum(int a, int b, int c) {
    return max(max(a, b), c); // Use built-in max function for clarity
}

__global__ void needle_cuda_shared_1(int* reference, int* matrix_cuda, int cols, int penalty, int i, int block_width) {
    int bx = blockIdx.x;
    int tx = threadIdx.x;

    int b_index_x = bx;
    int b_index_y = i - 1 - bx;

    int index = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + tx + (cols + 1);
    int index_n = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + tx + 1;
    int index_w = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x + cols;
    int index_nw = cols * BLOCK_SIZE * b_index_y + BLOCK_SIZE * b_index_x;

    __shared__ int temp[BLOCK_SIZE + 1][BLOCK_SIZE + 1];
    __shared__ int ref[BLOCK_SIZE][BLOCK_SIZE];

    // Prefetch reference values into shared memory
    for (int ty = 0; ty < BLOCK_SIZE; ty++) {
        ref[ty][tx] = reference[index + cols * ty];
    }
    __syncthreads();

    if (tx == 0) {
        temp[0][0] = matrix_cuda[index_nw];
    }
    temp[tx + 1][0] = matrix_cuda[index_w + cols * tx];
    temp[0][tx + 1] = matrix_cuda[index_n];
    __syncthreads();

    // Compute upper triangle of the dependency matrix in parallel
    for (int m = 0; m < BLOCK_SIZE; m++) {
        if (tx <= m) {
            int t_index_x = tx + 1;
            int t_index_y = m - tx + 1;
            temp[t_index_y][t_index_x] = maximum(temp[t_index_y - 1][t_index_x - 1] + ref[t_index_y - 1][t_index_x - 1],
                                                 temp[t_index_y][t_index_x - 1] - penalty,
                                                 temp[t_index_y - 1][t_index_x] - penalty);
        }
        __syncthreads();
    }

    // Compute lower triangle of the dependency matrix in parallel
    for (int m = BLOCK_SIZE - 2; m >= 0; m--) {
        if (tx <= m) {
            int t_index_x = tx + BLOCK_SIZE - m;
            int t_index_y = BLOCK_SIZE - tx;
            temp[t_index_y][t_index_x] = maximum(temp[t_index_y - 1][t_index_x - 1] + ref[t_index_y - 1][t_index_x - 1],
                                                 temp[t_index_y][t_index_x - 1] - penalty,
                                                 temp[t_index_y - 1][t_index_x] - penalty);
        }
        __syncthreads();
    }

    // Write back the results to global memory
    for (int ty = 0; ty < BLOCK_SIZE; ty++) {
        matrix_cuda[index + ty * cols] = temp[ty + 1][tx + 1];
    }
}