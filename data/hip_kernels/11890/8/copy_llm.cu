#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy(int *dst, int *input_itemsets, int max_rows, int max_cols, int lb0, int lb1, int ub0, int ub1)
{
    // Calculate global row and column index
    int r = blockIdx.y * blockDim.y + threadIdx.y + lb0;
    int c = blockIdx.x * blockDim.x + threadIdx.x + lb1;

    // If indexes are out of bounds return immediately
    if (r >= ub0 || c >= ub1) return;

    // Calculate 1D index in the array and copy the value
    int idx = r * max_cols + c;
    dst[idx] = input_itemsets[idx];
}