#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtract_kernel(float *in1, float *in2, float *out)
{
    // Calculate global thread ID
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Precompute bounds for efficiency
    unsigned int max_x = c_Size.x;
    unsigned int max_y = c_Size.y;
    unsigned int max_z = c_Size.z;

    // Skip computations if the thread ID exceeds bounds
    if (i >= max_x || j >= max_y || k >= max_z)
        return;

    // Use faster integer arithmetic for indexing
    long int id = ((long int)k * max_y + j) * max_x + i;

    // Perform subtraction
    out[id] = in1[id] - in2[id];
}