#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_volume_to_constant(float * out, float value)
{
    // Calculate global index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Check boundary conditions with flattening to enhance memory access pattern
    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
        return;

    long int id = (k * c_Size.y + j) * c_Size.x + i;

    // Writing constant value directly, which can enhance write performance
    out[id] = value;
}