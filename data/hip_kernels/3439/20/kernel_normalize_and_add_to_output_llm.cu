#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_normalize_and_add_to_output(float * dev_vol_in, float * dev_vol_out, float * dev_accumulate_weights, float * dev_accumulate_values)
{
    // Calculate indices using more efficient arithmetic
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= c_volSize.x || j >= c_volSize.y || k >= c_volSize.z)
    {
        return;
    }

    // Linearize index calculation for better performance
    long int out_idx = i + (j + k * c_volSize.y) * c_volSize.x;

    float eps = 1e-6;

    // Use direct index access and conditional operator for conciseness
    if (c_normalize)
    {
        dev_vol_out[out_idx] = abs(dev_accumulate_weights[out_idx]) > eps ? 
                               dev_vol_in[out_idx] + (dev_accumulate_values[out_idx] / dev_accumulate_weights[out_idx]) : 
                               dev_vol_in[out_idx];
    }
    else
    {
        dev_vol_out[out_idx] = dev_vol_in[out_idx] + dev_accumulate_values[out_idx];
    }
}