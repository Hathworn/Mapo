#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gradient_and_subtract_kernel(float * in, float * grad_x, float * grad_y, float * grad_z)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Exit early if indices are out of bounds
    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
        return;

    long int id = (k * c_Size.y + j) * c_Size.x + i;
    
    // Only calculate if not on boundary
    if (i < (c_Size.x - 1)) {
        long int id_x = id + 1;
        grad_x[id] -= (in[id_x] - in[id]) / c_Spacing.x;
    }
    
    if (j < (c_Size.y - 1)) {
        long int id_y = id + c_Size.x;
        grad_y[id] -= (in[id_y] - in[id]) / c_Spacing.y;
    }
    
    if (k < (c_Size.z - 1)) {
        long int id_z = id + c_Size.x * c_Size.y;
        grad_z[id] -= (in[id_z] - in[id]) / c_Spacing.z;
    }
}