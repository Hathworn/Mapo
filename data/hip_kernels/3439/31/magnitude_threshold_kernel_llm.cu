#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void magnitude_threshold_kernel(float *grad_x, float *grad_y, float *grad_z, float gamma)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Exit if out of bounds
    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
        return;

    // Calculate unique index using existing approach.
    long int id = (k * c_Size.y + j) * c_Size.x + i;

    // Compute norm using local registers to minimize memory access
    float gx = grad_x[id];
    float gy = grad_y[id];
    float gz = grad_z[id];
    float norm = sqrt(gx * gx + gy * gy + gz * gz);

    if (norm > gamma)
    {
        float ratio = gamma / norm;
        // Perform scaled assignment in place
        grad_x[id] = gx * ratio;
        grad_y[id] = gy * ratio;
        grad_z[id] = gz * ratio;
    }
}