#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiply_by_beta_kernel(float *input, float *output, float beta)
{
    // Calculate a flat index and check bounds in a single step.
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x +
                      (blockIdx.y * blockDim.y + threadIdx.y) * c_Size.x +
                      (blockIdx.z * blockDim.z + threadIdx.z) * c_Size.x * c_Size.y;

    if (id < c_Size.x * c_Size.y * c_Size.z) {
        output[id] = input[id] * beta; // Perform the multiplication.
    }
}