#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c) {
    // Calculate unique thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    // Loop over elements with strided access
    for (int i = tid; i < N; i += stride) {
        c[i] = a[i] + b[i];
    }
}