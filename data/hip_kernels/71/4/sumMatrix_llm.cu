#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrix(float *A, float *B, float *C, int nx, int ny) {
    // Calculate 2D thread index
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Check bounds and compute linear index
    if(ix < nx && iy < ny) {
        unsigned int idx = ix + iy * nx;
        C[idx] = A[idx] + B[idx];
    }
}