#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c) {
    // Use local variable for intermediate result
    int sum = a + b;
    // Store the result to global memory
    *c = sum;
}