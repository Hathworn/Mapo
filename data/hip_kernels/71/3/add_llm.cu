#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c) {
// Use thread index for better parallelism
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if(tid < N) {
c[tid] = a[tid] + b[tid];
}
}