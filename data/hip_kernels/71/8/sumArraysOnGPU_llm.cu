#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, int n) {
    // Calculate unique global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure not accessing out of bound elements
    if (idx < n) {
        C[idx] = A[idx] + B[idx];
    }
}
```
