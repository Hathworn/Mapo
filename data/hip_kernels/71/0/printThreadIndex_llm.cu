#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void printThreadIndex(int *A, const int nx, const int ny) {
    // Calculate global thread index
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;

    // Check if the thread index is within bounds
    if (ix < nx && iy < ny) {
        printf("thread_id (%d, %d) block_id (%d, %d) coordinate (%d, %d) global index %2d ival %2d\n", 
               threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx, A[idx]);
    }
}