#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    // Calculate global thread index for handling larger data
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure operation is within bounds
    if(tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}