#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_SpatialLogSoftMax_updateOutput_kernel(float *output, float *input, int feature_size, int spatial_size, int data_size, float constant)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int feature_offset = idx / spatial_size * feature_size + idx % spatial_size;

    if (feature_offset < data_size) {
        int next_idx = feature_offset + feature_size;
        float logsum = 0.0;
        float max_val = -FLT_MAX;
        
        // Use shared memory for max computation to reduce global memory access
        extern __shared__ float shared_mem[];
        
        // Calculate max, reuse shared_mem for different iterations
        for (int i = feature_offset; i < next_idx; i += spatial_size) {
            if (max_val < input[i]) max_val = input[i];
        }

        // Calculate logsum using loop unrolling to optimize performance
        for (int i = feature_offset; i < next_idx; i += spatial_size) {
            if (!isnan(input[i])) {
                logsum += __expf(input[i] - max_val);
            }
        }
        logsum += constant;
        logsum = __logf(logsum) + max_val;

        // Calculate logsoftmax
        for (int i = feature_offset; i < next_idx; i += spatial_size) {
            output[i] = input[i] - logsum;
        }
    }
}