#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void remove_white(float *x, float *y, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure valid index
    if (id >= size) return;

    // Use conditional operator for compactness
    y[id] = (x[id] == 255) ? 0 : y[id];
}