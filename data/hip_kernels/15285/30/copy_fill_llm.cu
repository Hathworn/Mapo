#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128

#define DISP_MAX 256

__global__ void copy_fill(float *in, float *out, int size, int in_size2, int in_size3, int out_size2, int out_size3)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        int out_x = id % out_size3;
        int out_y = id / out_size3;

        // Calculate in_x and in_y once to reduce operation counts
        int in_x_offset = (out_size3 - in_size3) / 2;
        int in_y_offset = (out_size2 - in_size2) / 2;

        // Use clamp to handle boundaries and simplify logic
        int x = min(in_size3 - 1, max(0, out_x - in_x_offset));
        int y = min(in_size2 - 1, max(0, out_y - in_y_offset));

        // Directly perform the memory copy using calculated indices
        out[id] = in[y * in_size3 + x];
    }
}