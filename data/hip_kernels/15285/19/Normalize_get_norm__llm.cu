#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void Normalize_get_norm_(float *input, float *norm, int size1, int size23, int size023)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size023) {
        int dim23 = id % size23;
        int dim0 = id / size23;

        float sum = 0.0;

        // Utilize loop unrolling for efficiency
        int dim1;
        for (dim1 = 0; dim1 < size1 - 3; dim1 += 4) {
            float x0 = input[(dim0 * size1 + dim1) * size23 + dim23];
            float x1 = input[(dim0 * size1 + dim1 + 1) * size23 + dim23];
            float x2 = input[(dim0 * size1 + dim1 + 2) * size23 + dim23];
            float x3 = input[(dim0 * size1 + dim1 + 3) * size23 + dim23];
            sum += x0 * x0 + x1 * x1 + x2 * x2 + x3 * x3;
        }
        // Handle remaining iterations
        for (; dim1 < size1; dim1++) {
            float x = input[(dim0 * size1 + dim1) * size23 + dim23];
            sum += x * x;
        }

        norm[dim0 * size23 + dim23] = sum + 1e-5;
    }
}