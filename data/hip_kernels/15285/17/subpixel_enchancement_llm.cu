#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void subpixel_enchancement(float *d0, float *c2, float *out, int size, int dim23, int disp_max) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Process only if within valid range
    if (id >= size) return;

    int d = d0[id];
    out[id] = d;

    // Optimize memory access by evaluating condition first
    if (1 <= d && d < disp_max - 1) {
        float cn = c2[(d - 1) * dim23 + id];
        float cz = c2[d * dim23 + id];
        float cp = c2[(d + 1) * dim23 + id];
        
        // Combine computations to reduce repeated operations
        float denom = cp + cn - 2 * cz;
        
        if (denom > 5e-6) { // Slightly adjust threshold for numerical stability
            out[id] = d - min(1.0f, max(-1.0f, 0.5f * (cp - cn) / denom));
        }
    }
}