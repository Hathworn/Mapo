#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void rho(float *x, int size, float lambda)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Process multiple elements if out of bounds
    for (int i = id; i < size; i += stride) {
        x[i] = 1 - exp(-x[i] / lambda);
    }
}