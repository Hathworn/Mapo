#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_SpatialLogSoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int feature_size, int spatial_size, int data_size)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int offset = (idx / spatial_size) * feature_size + idx % spatial_size;

    if (offset < data_size) {
        int next_idx = offset + feature_size;
        float gradSum = 0.0f;

        // Unrolling the loop to improve memory access efficiency
        #pragma unroll
        for (int i = offset; i < next_idx; i += spatial_size) {
            gradSum += gradOutput[i];
        }

        // Updating gradient input using precomputed gradSum
        #pragma unroll
        for (int i = offset; i < next_idx; i += spatial_size) {
            gradInput[i] = gradOutput[i] - __expf(output[i]) * gradSum;
        }
    }
}