#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void remove_occluded(float *y, int size, int size3)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        int x = id % size3;

        // Preload the current y[id] value
        float y_id_val = y[id];

        for (int i = 1; x + i < size3; i++) {
            // Use a temporary variable to avoid repetitive memory access
            float y_next_val = y[id + i];
            if (i - y_next_val < -y_id_val) {
                y[id] = 0;
                break;
            }
        }
    }
}