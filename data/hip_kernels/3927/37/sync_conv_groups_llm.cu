#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize with threadfence and syncthreads if necessary
__global__ void sync_conv_groups() {
    __threadfence(); // Ensure all writes are visible
    __syncthreads(); // Synchronize all threads in the block
}