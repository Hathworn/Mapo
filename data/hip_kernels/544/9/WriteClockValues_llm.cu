#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void WriteClockValues(unsigned int *completionTimes, unsigned int *threadIDs)
{
    // Pre-compute values outside the loop
    size_t blockId = blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z);
    size_t threadId = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z);

    size_t totalBlockSize = blockDim.x * blockDim.y * blockDim.z;
    size_t globalIndex = blockId * totalBlockSize + threadId;

    // Assign the current clock cycle to the globalIndex position
    completionTimes[globalIndex] = clock(); 

    // Compact thread ID calculation
    threadIDs[globalIndex] = (threadIdx.y << 4) | threadIdx.x;
}