#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Return32( int *sum, int *out, const int *pIn )
{
    // Calculate a unique global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use idx rather than threadIdx.x for better scalability
    out[idx] = atomicAdd(&sum[idx], *pIn);
}