#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void p2pPingPongLatencyTest(void *_pLocal, void *_pRemote, uint64_t *pTimestamps, int bWait, int cIterations)
{
    volatile int *pLocal = (volatile int *)_pLocal;
    volatile int *pRemote = (volatile int *)_pRemote;
    int pingpongValue = 0;

    // Utilize shared memory for common variables
    __shared__ int sharedPingpongValue;
    if (threadIdx.x == 0) {
        sharedPingpongValue = pingpongValue;
    }
    __syncthreads();

    while (cIterations--) {
        *pTimestamps++ = clock64();

        if (bWait) {
            while (*pLocal != sharedPingpongValue) {
                // Use __threadfence() to prevent stale cache reads
                __threadfence();
            }
        }
        bWait = 1;

        // Update the shared variable which prevents redundant assignments to pingpongValue
        sharedPingpongValue = 1 - sharedPingpongValue;

        if (threadIdx.x == 0) {
            *pRemote = sharedPingpongValue;
        }
        __syncthreads();
    }
}