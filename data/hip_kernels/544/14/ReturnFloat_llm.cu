#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReturnFloat(float *sum, float *out, const float *pIn)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global index
    atomicAdd(&out[idx], pIn[idx]);  // Optimize index calculation and atomicAdd
}