#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReturnFloat(float *sum, float *out, const float *pIn) {
    extern __shared__ float s[];
    int tid = threadIdx.x;
    s[tid] = pIn[tid];
    __syncthreads();

    // Use atomic operations directly on output to avoid shared memory contention
    atomicAdd(&sum[tid], pIn[tid]);
    __syncthreads();

    // Write result directly to output
    out[tid] = sum[tid];
}