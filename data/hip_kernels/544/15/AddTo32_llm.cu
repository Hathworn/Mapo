#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AddTo32(int *sum, int *out, const int *pIn)
{
    // Optimize for parallel execution
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    atomicAdd(&out[idx], *pIn);
}