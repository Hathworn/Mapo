#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Return32( int *sum, int *out, const int *pIn )
{
    extern __shared__ int s[];

    // Load input into shared memory
    int idx = threadIdx.x;
    s[idx] = pIn[idx];
    __syncthreads();

    // Use shared memory for atomic operations
    atomicAdd(&s[idx], *sum); // Assuming a single accumulation target, using *sum
    __syncthreads();

    // Write results back to output
    out[idx] = s[idx];
}