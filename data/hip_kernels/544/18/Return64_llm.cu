#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Return64(unsigned long long *sum, unsigned long long *out, const unsigned long long *pIn)
{
    // Optimize by reducing atomic operation scope to minimal required index
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned long long val = (idx < blockDim.x) ? *pIn : 0; // Ensure valid read within bounds
    out[idx] = atomicAdd(&sum[idx], val); // Perform atomicAdd only if `idx` is valid
}