#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AddToFloat(float* sum, float* out, const float* pIn)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    (void)atomicAdd(&out[idx], pIn[idx]); // Use global thread index for memory access
}