#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RadixHistogram_device(int *dptrHistogram, const int *in, size_t N, int shift, int mask)
{
    extern __shared__ int sharedHistogram[];
    
    for (int i = threadIdx.x; i < (1 << mask); i += blockDim.x) {
        sharedHistogram[i] = 0; // Initialize shared memory
    }
    __syncthreads();

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        int index = (in[i] & mask) >> shift;
        atomicAdd(&sharedHistogram[index], 1); // Use shared memory
    }
    __syncthreads();

    for (int i = threadIdx.x; i < (1 << mask); i += blockDim.x) {
        if (sharedHistogram[i] > 0) {
            atomicAdd(dptrHistogram + i, sharedHistogram[i]); // Write to global memory
        }
    }
}