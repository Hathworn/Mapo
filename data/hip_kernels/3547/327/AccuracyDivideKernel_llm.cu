#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Each thread block processes the division independently, assuming one block.
    if (threadIdx.x == 0) { // Use only the first thread for division to avoid race conditions
        *accuracy /= N;
    }
}