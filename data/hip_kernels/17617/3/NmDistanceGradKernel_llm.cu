#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NmDistanceGradKernel(int b, int n, const float* xyz1, int m, const float* xyz2, const float* grad_dist1, const int* idx1, float* grad_xyz1, float* grad_xyz2) {
    // Calculate unique thread index within a block
    int batchId = blockIdx.x;
    int threadId = threadIdx.x + blockIdx.y * blockDim.x;

    for (int i = batchId; i < b; i += gridDim.x) {
        for (int j = threadId; j < n; j += blockDim.x * gridDim.y) {
            // Precompute indices and load values into registers
            int index1 = (i * n + j) * 3;
            float x1 = xyz1[index1];
            float y1 = xyz1[index1 + 1];
            float z1 = xyz1[index1 + 2];
            int j2 = idx1[i * n + j];
            int index2 = (i * m + j2) * 3;
            float x2 = xyz2[index2];
            float y2 = xyz2[index2 + 1];
            float z2 = xyz2[index2 + 2];
            float g = grad_dist1[i * n + j] * 2;

            // Use register values and reduce atomic operations
            float dx = g * (x1 - x2);
            float dy = g * (y1 - y2);
            float dz = g * (z1 - z2);

            atomicAdd(&grad_xyz1[index1], dx);
            atomicAdd(&grad_xyz1[index1 + 1], dy);
            atomicAdd(&grad_xyz1[index1 + 2], dz);
            atomicAdd(&grad_xyz2[index2], -dx);
            atomicAdd(&grad_xyz2[index2 + 1], -dy);
            atomicAdd(&grad_xyz2[index2 + 2], -dz);
        }
    }
}