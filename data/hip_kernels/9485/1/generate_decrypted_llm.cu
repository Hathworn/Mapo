#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_decrypted(int *pDataPointer, int *pRandomData, int *pEncryptedData, long long int pSize) 
{
    long long int index = blockIdx.x * blockDim.x + threadIdx.x;
    long long int numElements = pSize / sizeof(int);

    if (index < numElements) // Use < for better readability and to avoid boundary issues
    {
        pEncryptedData[index] = pDataPointer[index] ^ pRandomData[index]; // Use array syntax for clarity
    }
}