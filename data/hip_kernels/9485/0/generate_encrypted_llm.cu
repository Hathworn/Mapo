#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_encrypted(int *pDataPointer, int *pRandomData, int *pEncryptedData, long long int pSize)
{
    long long int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Simplified condition to avoid redundant return statement
    if (index < (pSize / sizeof(int)))
    {
        // Perform encryption using XOR
        pEncryptedData[index] = pDataPointer[index] ^ pRandomData[index];
    }
}