#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReductionMin(unsigned int *sdata, unsigned int *results, int n) {
    unsigned int tx = threadIdx.x;

    // Use unrolling for efficient reduction
    for (unsigned int offset = blockDim.x >> 1; offset > 32; offset >>= 1) {
        __syncthreads();
        if (tx < offset) {
            if (sdata[tx + offset] < sdata[tx] || sdata[tx] == 0) {
                sdata[tx] = sdata[tx + offset];
            }
        }
    }

    // Unroll the last warp
    if (tx < 32) {
        volatile unsigned int* vsmem = sdata;
        if (vsmem[tx + 32] < vsmem[tx] || vsmem[tx] == 0) vsmem[tx] = vsmem[tx + 32];
        if (vsmem[tx + 16] < vsmem[tx] || vsmem[tx] == 0) vsmem[tx] = vsmem[tx + 16];
        if (vsmem[tx + 8] < vsmem[tx] || vsmem[tx] == 0) vsmem[tx] = vsmem[tx + 8];
        if (vsmem[tx + 4] < vsmem[tx] || vsmem[tx] == 0) vsmem[tx] = vsmem[tx + 4];
        if (vsmem[tx + 2] < vsmem[tx] || vsmem[tx] == 0) vsmem[tx] = vsmem[tx + 2];
        if (vsmem[tx + 1] < vsmem[tx] || vsmem[tx] == 0) vsmem[tx] = vsmem[tx + 1];
    }

    // Thread 0 writes the result
    if (threadIdx.x == 0) {
        *results = sdata[0];
    }
}