#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BitonicMergeSort(float *d_output, float *d_input, int subarray_size) 
{
    extern __shared__ float shared_data[];

    // Internal index for sorting of the subarray
    int index = threadIdx.x;
    int index_global = index + blockDim.x * blockIdx.x;

    // Calculate the number of portions
    int portions = __float2int_rd(log2f((float)subarray_size)) - 1;

    // Copying of data portion dedicated to this block into shared memory
    shared_data[index] = d_input[index_global];
    __syncthreads();

    // Perform bitonic sort
    for (int portion = 0; portion <= portions; portion++) 
    {
        int offset = 1 << portion;
        int threads_in_box = offset << 1;  // Calculate at the beginning of each portion
        int boxI = threadIdx.x / threads_in_box;

        for (int subportion = portion; subportion >= 0; subportion--) 
        {
            offset = 1 << subportion;
            threads_in_box = offset << 1;
            int arrow_bottom = index % threads_in_box;

            float temp = shared_data[index];

            if (((boxI + 1) % 2) == 1) 
            {
                // Top down
                if (arrow_bottom < offset && shared_data[index + offset] < temp) 
                {
                    shared_data[index] = shared_data[index + offset];
                    shared_data[index + offset] = temp;
                }
            } 
            else 
            {
                // Bottom up
                if (arrow_bottom >= offset && shared_data[index - offset] < temp) 
                {
                    shared_data[index] = shared_data[index - offset];
                    shared_data[index - offset] = temp;
                }
            }
            __syncthreads();  // Synchronize threads at the end of each subportion
        }
    }

    // Store the sorted subarray back to global memory
    d_output[index_global] = shared_data[index];
}
```  
