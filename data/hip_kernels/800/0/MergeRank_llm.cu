#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MergeRank(float * d_input, float * d_output)
{
    // Using shared memory to store input values for faster access
    __shared__ float shared_input[4096];

    int indexA = blockIdx.x * blockDim.x + threadIdx.x;
    int indexB = indexA + 2048;

    // Loading inputs to shared memory
    shared_input[threadIdx.x] = d_input[indexA];
    shared_input[threadIdx.x + 2048] = d_input[indexB];
    __syncthreads();

    float temp1 = shared_input[threadIdx.x];
    float temp2 = shared_input[threadIdx.x + 2048];

    int rank1 = 2048;
    int rank2 = 0;

    // Parallel binary search for better performance
    for (int stride = 1024; stride > 0; stride >>= 1) {
        int pos = rank1 - stride;
        if (pos >= 0 && shared_input[pos] < temp1) {
            rank1 = pos;
        }

        pos = rank2 + stride;
        if (pos < 2048 && shared_input[pos] < temp2) {
            rank2 = pos;
        }
    }

    __syncthreads();
    d_output[indexA + rank1 + 1] = temp1;
    d_output[indexB + rank2 + 1] = temp2;
}