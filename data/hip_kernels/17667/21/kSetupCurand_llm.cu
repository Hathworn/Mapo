#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Optimize kernel launch: process only active threads
    if (tidx < NUM_RND_THREADS_PER_BLOCK * gridDim.x) {
        // Each thread gets unique sequence number for variance
        hiprand_init(seed, tidx, 0, &state[tidx]);
    }
}