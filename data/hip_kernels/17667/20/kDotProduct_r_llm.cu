#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DP_BLOCKSIZE 512  // Ensure DP_BLOCKSIZE is defined appropriately

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numCols, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float tempSum = 0.0f;  // Use register for intermediate sum

    while (eidx < numElements) {  // Loop to cover all elements within block
        tempSum += a[eidx] * b[eidx];
        eidx += numCols;
    }
    shmem[threadIdx.x] = tempSum;
    __syncthreads();

    // Optimize reduction using sequential addressing
    for (uint stride = DP_BLOCKSIZE / 2; stride > 32; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Unroll last warp
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}