#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate the global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;  // Calculate the stride

    // Loop over the target array in steps of stride
    for (uint i = idx; i < tgtWidth * tgtHeight; i += stride) {
        const uint y = i / tgtWidth;  // Calculate y-coordinate in target
        const uint x = i % tgtWidth;  // Calculate x-coordinate in target
        const uint srcY = y % srcHeight;  // Wrap y-coordinate within source height
        const uint srcX = x % srcWidth;  // Wrap x-coordinate within source width
        tgt[i] = src[srcY * srcWidth + srcX];  // Tile the source into target
    }
}