#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_DIM 16 // Ensure TILE_DIM is defined

__global__ void gpu_multiply(float* A, float* B, float* C, int ARows, int ACols, int BRows, int BCols, int CRows, int CCols) {

    float CValue = 0;

    int Row = blockIdx.y * TILE_DIM + threadIdx.y;
    int Col = blockIdx.x * TILE_DIM + threadIdx.x;

    __shared__ float As[TILE_DIM][TILE_DIM];
    __shared__ float Bs[TILE_DIM][TILE_DIM];

    // Optimize the loop by reducing condition checks
    for (int k = 0; k < (ACols + TILE_DIM - 1) / TILE_DIM; k++) {

        // Load tile from A matrix
        int tiledACol = k * TILE_DIM + threadIdx.x;
        As[threadIdx.y][threadIdx.x] = (tiledACol < ACols && Row < ARows) ? A[Row * ACols + tiledACol] : 0.0f;

        // Load tile from B matrix
        int tiledBRow = k * TILE_DIM + threadIdx.y;
        Bs[threadIdx.y][threadIdx.x] = (tiledBRow < BRows && Col < BCols) ? B[tiledBRow * BCols + Col] : 0.0f;

        __syncthreads();

        for (int n = 0; n < TILE_DIM; ++n)
            CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

        __syncthreads();
    }

    // Write the computed value to C matrix
    if (Row < CRows && Col < CCols) 
        C[Row * CCols + Col] = CValue;
}