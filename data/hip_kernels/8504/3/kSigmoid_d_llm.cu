#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSigmoid_d(const int nThreads, const float *input, float *output) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < nThreads) { // Check bounds within the kernel
        output[i] = input[i] * (1 - input[i]);
    }
}