#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_add(float* first, float* second, size_t sizeFirst)
{
    // Calculate global thread index
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;

    // Use stride loop pattern for better memory utilization
    for (; threadId < sizeFirst; threadId += blockDim.x * gridDim.x) {
        first[threadId] += second[threadId];
    }
}