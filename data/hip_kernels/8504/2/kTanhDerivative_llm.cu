#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTanhDerivative(const int nThreads, float const *input, float *output)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Utilize strided loop pattern for better memory access
    for (; i < nThreads; i += blockDim.x * gridDim.x)
    {
        float val = tanh(input[i]);
        output[i] = 1 - (val * val);
    }
}