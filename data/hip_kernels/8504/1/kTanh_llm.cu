#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTanh(const int nThreads, float const *input, float *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Optimize loop with stride increment for better coalesced memory access
    for (int i = idx; i < nThreads; i += stride) {
        output[i] = tanh(input[i]);
    }
}