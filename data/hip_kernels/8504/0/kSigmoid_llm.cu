#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kSigmoid kernel function using shared memory
__global__ void kSigmoid(const int nThreads, const float *input, float *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global thread index
    if (idx < nThreads) { // Ensure idx is within the valid range
        output[idx] = 1.0f / (1.0f + expf(-input[idx])); // Use expf for better performance
    }
}