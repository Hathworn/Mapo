#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_mull2(float* a, float* b, float* c, int n, int m, int p)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Update block size usage
    int j = blockIdx.y;

    if (i < n) { // Boundary check for i
        float sum = 0.0f; 
        for (int k = 0; k < p; ++k) {
            sum += b[i + n * k] * c[k + p * j];
        }
        a[i + n * j] = sum;
    }
}