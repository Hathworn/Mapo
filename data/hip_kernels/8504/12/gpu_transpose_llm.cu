#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_transpose(const float* src, float* dst, int colssrc, int colsdst, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int strd = blockDim.x * gridDim.x;

    for (int tid = idx; tid < n; tid += strd) { // Use loop increments by stride
        int cdst = tid % colsdst;
        int rdst = tid / colsdst;
        int rsrc = cdst;
        int csrc = rdst;
        dst[tid] = src[rsrc * colssrc + csrc];
    }
}