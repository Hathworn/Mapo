#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use loop to ensure all threads participate and process multiple elements
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        ingrad[i] = indata[i] > 0 ? outgrad[i] : 0; // Simplify condition and calculation
    }
}