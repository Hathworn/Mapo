#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use loop unrolling and stride to improve memory access patterns
    for (int i = tid; i < N; i += stride) {
        float input_val = in[i]; // Load once
        out[i] = input_val > 0 ? input_val : 0; // Conditional store
    }
}