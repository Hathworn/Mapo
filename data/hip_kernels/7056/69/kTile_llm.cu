#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Use shared memory for better cache performance
    extern __shared__ float sharedSrc[];
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    
    // Load src into shared memory
    if (idx < srcWidth * srcHeight) {
        sharedSrc[idx] = src[idx];
    }
    __syncthreads();

    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        tgt[i] = sharedSrc[srcY * srcWidth + srcX];
    }
}