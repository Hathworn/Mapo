#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeSphereVertexDistancesKernel(float4 *V, float *dist, unsigned int *NEIGHBOR, unsigned int *NBOFFSETS, unsigned int *nNeighbors, unsigned int nVertices, float circumference)
{
    int n, N;
    int offset, soffset;

    // Calculate the vertex index using thread and block indices
    int vidxb = 4 * (blockIdx.x * blockDim.x) + threadIdx.x;
    int basevert = 4 * (blockIdx.x * blockDim.x);

    int vidx, tab;
    float4 nv, tv;
    float dot, n1, n2, norm;

    // Shared memory cache for vertices
    __shared__ float4 SI[4 * BLOCK_SIZE_CVD];

    // Thread index in block
    int bidx = threadIdx.x;
    // Load vertices into shared memory
    for (vidx = vidxb; vidx < vidxb + 4 * BLOCK_SIZE_CVD; vidx += BLOCK_SIZE_CVD)
    {
        if (vidx < nVertices)
        {
            SI[bidx] = V[vidx];
            bidx += BLOCK_SIZE_CVD;
        }
    }

    __syncthreads();

    bidx = threadIdx.x;
    // Process the vertices
    for (vidx = vidxb; vidx < vidxb + 4 * BLOCK_SIZE_CVD; vidx += BLOCK_SIZE_CVD)
    {
        if (vidx < nVertices)
        {
            offset = NBOFFSETS[vidx];
            N = nNeighbors[vidx];
            tv = SI[bidx];

            bidx += BLOCK_SIZE_CVD;

            for (n = 0; n < N; n++)
            {
                soffset = NEIGHBOR[offset + n];

                // Optimized neighbor vertex retrieval
                tab = soffset - basevert;
                nv = (tab >= 0 && tab < 4 * BLOCK_SIZE_CVD) ? SI[tab] : V[soffset];

                // Use FMA intrinsics to compute dot product and magnitudes
                dot = __fmul_rn(tv.x, nv.x);
                dot = __fadd_rn(dot, __fmul_rn(tv.y, nv.y));
                dot = __fadd_rn(dot, __fmul_rn(tv.z, nv.z));

                n1 = __fmul_rn(tv.x, tv.x);
                n1 = __fadd_rn(n1, __fmul_rn(tv.y, tv.y));
                n1 = __fadd_rn(n1, __fmul_rn(tv.z, tv.z));

                n2 = __fmul_rn(nv.x, nv.x);
                n2 = __fadd_rn(n2, __fmul_rn(nv.y, nv.y));
                n2 = __fadd_rn(n2, __fmul_rn(nv.z, nv.z));

                norm = __fmul_rn(__fsqrt_rn(n1), __fsqrt_rn(n2));

                // Numerical stability checks
                if (norm < 1.0e-7f || fabsf(dot) > norm)
                {
                    dist[offset + n] = 0.0f;
                }
                else
                {
                    dist[offset + n] = __fmul_rn(circumference, fabsf(acosf(dot / norm)));
                }
            }
        }
    }
}