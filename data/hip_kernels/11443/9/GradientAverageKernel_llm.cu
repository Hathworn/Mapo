#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GradientAverageKernel(float4 *D, float4 *TD, unsigned int *NEIGHBOR, unsigned int *NBOFFSETS, unsigned int *nNeighbors, unsigned int nVertices)
{
    // Calculate global vertex index
    int vidxb = 4 * (blockIdx.x * blockDim.x) + threadIdx.x;

    // Shared memory for caching data
    __shared__ float4 SI[4 * BLOCK_SIZE_AVGG];

    int bidx = 4 * threadIdx.x;

    // Cache vertices data into shared memory
    for (int vidx = vidxb; vidx < vidxb + 4 * BLOCK_SIZE_AVGG; vidx += BLOCK_SIZE_AVGG)
    {
        if (vidx < nVertices)
        {
            SI[bidx] = D[vidx];
            bidx++;
        }
    }

    __syncthreads();

    bidx = 4 * threadIdx.x;

    // Process each vertex in the block
    for (int vidx = vidxb; vidx < vidxb + 4 * BLOCK_SIZE_AVGG; vidx += BLOCK_SIZE_AVGG)
    {
        if (vidx < nVertices)
        {
            int offset = NBOFFSETS[vidx];
            int N = nNeighbors[vidx];

            float4 td = SI[bidx++];
            float4 nbd;

            // Accumulate neighbors data
            for (int n = 0; n < N; n++)
            {
                int soffset = NEIGHBOR[offset + n];
                if (soffset < nVertices) // Ensure valid access memory
                {
                    nbd = D[soffset];
                    td.x += nbd.x;
                    td.y += nbd.y;
                    td.z += nbd.z;
                }
            }

            // Compute average
            float invNplus1 = 1.0f / (float)(N + 1);
            td.x *= invNplus1;
            td.y *= invNplus1;
            td.z *= invNplus1;

            // Store result
            TD[vidx] = td;
        }
    }
}