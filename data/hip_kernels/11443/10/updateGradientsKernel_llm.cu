#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateGradientsKernel(float4 *D, float4 *TD, unsigned int nVertices)
{
    int vidx = 4 * (blockIdx.x * blockDim.x) + threadIdx.x;
    
    // Unroll the loop to improve performance
    if (vidx < nVertices) {
        D[vidx] = TD[vidx];
        if (vidx + BLOCK_SIZE_AVGG < nVertices) 
            D[vidx + BLOCK_SIZE_AVGG] = TD[vidx + BLOCK_SIZE_AVGG];
        if (vidx + 2 * BLOCK_SIZE_AVGG < nVertices) 
            D[vidx + 2 * BLOCK_SIZE_AVGG] = TD[vidx + 2 * BLOCK_SIZE_AVGG];
        if (vidx + 3 * BLOCK_SIZE_AVGG < nVertices) 
            D[vidx + 3 * BLOCK_SIZE_AVGG] = TD[vidx + 3 * BLOCK_SIZE_AVGG];
    }
}