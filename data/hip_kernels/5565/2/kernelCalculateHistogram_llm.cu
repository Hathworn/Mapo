#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelCalculateHistogram(unsigned int* histogram, unsigned char* rawPixels, long chunkSize, long totalPixels)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;  // Calculate stride

    for (int i = id * chunkSize; i < totalPixels; i += stride * chunkSize) {
        int end = min(i + chunkSize, totalPixels);  // Calculate end to prevent overstepping
        for (int j = i; j < end; ++j) {
            int pixelValue = (int)rawPixels[j];
            atomicAdd(&histogram[pixelValue], 1);
        }
    }
}