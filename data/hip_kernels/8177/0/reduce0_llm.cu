#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void reduce0(float* g_odata, float* g_idata1, float* g_idata2) {
    extern __shared__ float sdata[];

    // Calculate global index
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Bounds check
    if (i >= gridDim.x * blockDim.x) return;

    // Load from global memory to shared memory
    sdata[tid] = g_idata1[i] * g_idata2[i];
    __syncthreads();

    // Perform tree-based reduction
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}