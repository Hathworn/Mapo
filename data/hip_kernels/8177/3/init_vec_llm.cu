#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// The __restrict__ qualifier hints to the compiler that vec does not alias 
__global__ void init_vec(float* __restrict__ vec, float value) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < gridDim.x * blockDim.x) { // Check to avoid out-of-bounds memory write
        vec[tid] = value;
    }
}