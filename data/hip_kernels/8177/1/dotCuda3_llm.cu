#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void dotCuda3(float *a, float *b, float *c) {
    __shared__ float cache[1024];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int cacheIndex = threadIdx.x;
    
    // Efficiently load data into shared memory
    float temp = 0.0f;
    if (tid < gridDim.x * blockDim.x) {
        temp = a[tid] * b[tid];
    }
    cache[cacheIndex] = temp;
    __syncthreads();
    
    // Optimize reduction with unrolling and warp shuffle
    for (unsigned int i = blockDim.x >> 1; i > 0; i >>= 1) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }
    
    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}