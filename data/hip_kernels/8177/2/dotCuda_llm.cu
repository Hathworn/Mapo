#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void dotCuda(float* tmp, float* t1, float* t2, int size) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    // Calculate partial dot product
    for (int idx = i; idx < size; idx += stride) {
        tmp[idx] = t1[idx] * t2[idx];
    }
    __syncthreads();

    // Reduce step: Use shared memory to accumulate results
    extern __shared__ float sdata[];
    sdata[threadIdx.x] = (i < size) ? tmp[i] : 0;
    __syncthreads();

    // Reduce within block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        tmp[blockIdx.x] = sdata[0];
    }
}
