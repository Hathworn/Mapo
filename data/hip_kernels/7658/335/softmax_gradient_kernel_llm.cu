#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    // Initialize local accumulation variable
    float local_sum = 0.0f;

    // Perform partial reduction in a loop using thread-stride
    for (int i = idx; i < dim; i += blockDim.x) {
        local_sum += dY[i] * Y[i];
    }

    // Store partial results in shared memory
    reduction_buffer[idx] = local_sum;
    __syncthreads();

    // Reduce within the block to form the total inner product
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }

    // Broadcasting the final result to all threads
    float inner_product = reduction_buffer[0];

    // Compute gradient
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - inner_product);
    }
}