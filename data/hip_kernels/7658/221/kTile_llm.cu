#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    const unsigned int numEls = tgtWidth * tgtHeight;
    for (uint i = idx; i < numEls; i += numThreads) {
        // Calculate target and source indices
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Direct assignment from source to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}