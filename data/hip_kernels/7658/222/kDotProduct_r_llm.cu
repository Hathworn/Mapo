#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Loop unrolling to improve memory access coalescing
    if (eidx < gridDim.x * DP_BLOCKSIZE) {
        for (; eidx + 3 < numElements; eidx += gridDim.x * DP_BLOCKSIZE * 4) {
            sum += a[eidx] * b[eidx];
            sum += a[eidx + 1] * b[eidx + 1];
            sum += a[eidx + 2] * b[eidx + 2];
            sum += a[eidx + 3] * b[eidx + 3];
        }
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            sum += a[eidx] * b[eidx];
        }
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Use warp reduction to minimize synchronization
    if (threadIdx.x < 128) { shmem[threadIdx.x] += shmem[threadIdx.x + 128]; }
    __syncthreads();
    if (threadIdx.x < 64) { shmem[threadIdx.x] += shmem[threadIdx.x + 64]; }
    __syncthreads();

    if (threadIdx.x < 32) {
        volatile float* mysh = &shmem[threadIdx.x];
        *mysh += mysh[32];
        *mysh += mysh[16];
        *mysh += mysh[8];
        *mysh += mysh[4];
        *mysh += mysh[2];
        *mysh += mysh[1];
    }

    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}