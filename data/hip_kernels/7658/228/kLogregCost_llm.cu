#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Efficiently check if label probability is max; initialize outside loop
        int isCorrect = (labelp == maxp) ? 1 : 0;
        
        // Count number of maximum probabilities
        int numMax = 0;
        for (int i = 0; i < numOut; i++) {
            // Increment inside the condition
            numMax += (probs[i * numCases + tx] == maxp);
        }
        
        // Use ternary operator for concise assignment
        correctProbs[tx] = isCorrect ? 1.0f / float(numMax) : 0.0f;
    }
}