#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    // Use blockDim.x for thread calculations to enhance performance
    hiprand_init(seed, tidx, 0, &state[tidx]);
}