#pragma once
#include <hip/hip_runtime.h>


#  define CUDA_SAFE_CALL_NO_SYNC( call) {                                    \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(EXIT_FAILURE);                                                  \
    } }

#  define CUDA_SAFE_CALL( call)     CUDA_SAFE_CALL_NO_SYNC(call);                                            \

#  define CUDA_SAFE_THREAD_SYNC( ) {                                         \
    hipError_t err = CUT_DEVICE_SYNCHRONIZE();                                 \
    if ( hipSuccess != err) {                                               \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
    } }

// from http://forums.nvidia.com/index.php?showtopic=186669
static __device__ uint get_smid(void) {
     uint ret;
     asm("mov.u32 %0, %smid;" : "=r"(ret) );
     return ret;
}
