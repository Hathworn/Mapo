#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_kernel_texture_2d(unsigned char *surface, int width, int height, size_t pitch, float t) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure threads only process valid pixels
    if (x < width && y < height) {
        // Utilize shared memory for pixel computation if possible
        
        // Get pointer to pixel at (x, y)
        float *pixel = (float *)(surface + y * pitch) + 4 * x;

        // Calculate values for red and green components
        float normalized_x = (2.0f * x) / width - 1.0f;
        float normalized_y = (2.0f * y) / height - 1.0f;
        float value_x = 0.5f + 0.5f * cos(t + 10.0f * normalized_x);
        float value_y = 0.5f + 0.5f * cos(t + 10.0f * normalized_y);

        // Populate pixel value
        float pow_value_x = value_x * value_x * value_x;
        float pow_value_y = value_y * value_y * value_y;
        pixel[0] = 0.5f * pixel[0] + 0.5f * pow_value_x; // red
        pixel[1] = 0.5f * pixel[1] + 0.5f * pow_value_y; // green
        pixel[2] = 0.5f + 0.5f * cos(t); // blue
        pixel[3] = 1.0f; // alpha
    }
}