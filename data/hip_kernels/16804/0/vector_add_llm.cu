#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float* out, float* a, float* b, int n) {
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Process all elements each thread is responsible for
    for (int i = index; i < n; i += blockDim.x * gridDim.x) {
        out[i] = a[i] + b[i];
    }
}