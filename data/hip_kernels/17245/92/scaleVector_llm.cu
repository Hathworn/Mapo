#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Use stride to enable each block to handle more elements
    for (int pos = blockIdx.x * blockDim.x + threadIdx.x; pos < len; pos += blockDim.x * gridDim.x)
    {
        d_res[pos] = d_src[pos] * scale;
    }
}