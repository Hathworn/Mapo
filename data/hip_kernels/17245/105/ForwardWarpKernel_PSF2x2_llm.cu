#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    // pixel containing bottom left corner
    float px;
    float py;
    float dx = modff (cx, &px);
    float dy = modff (cy, &py);
    // target pixel integer coords
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);
    float value = src[image_row_offset + j];

    #pragma unroll // Unroll the loop to eliminate branches
    for (int k = 0; k < 4; ++k)
    {
        float weight;
        // Define shifts in box coordinates
        int shift_x = (k == 1 || k == 2) ? -1 : 0;
        int shift_y = (k == 2 || k == 3) ? -1 : 0;

        // Update tx and ty accordingly
        int tmp_tx = tx + shift_x;
        int tmp_ty = ty + shift_y;

        if (!(tmp_tx >= w || tmp_tx < 0 || tmp_ty >= h || tmp_ty < 0))
        {
            // Calculate weight based on corner
            switch (k)
            {
                case 0: weight = dx * dy; break;            // bottom right
                case 1: weight = (1.0f - dx) * dy; break;   // bottom left
                case 2: weight = (1.0f - dx) * (1.0f - dy); break; // upper left
                case 3: weight = dx * (1.0f - dy); break;   // upper right
            }

            atomicAdd(dst + tmp_ty * image_stride + tmp_tx, value * weight);
            atomicAdd(normalization_factor + tmp_ty * image_stride + tmp_tx, weight);
        }
    }
}