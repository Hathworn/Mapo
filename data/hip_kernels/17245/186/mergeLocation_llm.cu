#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Load loc_ into registers to reduce global memory access
        short2 loc = __ldg(&loc_[ptidx]);

        // Compute scaled x and y values
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}