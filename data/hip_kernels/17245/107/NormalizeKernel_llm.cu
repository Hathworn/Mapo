#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = blockIdx.y;  // Use blockIdx.y to iterate over rows

    if (i >= h || j >= w) return;

    const int pos = i * s + j;

    float scale = normalization_factor[pos];

    float invScale = (scale == 0.0f) ? 1.0f : __fdividef(1.0f, scale);  // Use fast division

    image[pos] *= invScale;
}
```
