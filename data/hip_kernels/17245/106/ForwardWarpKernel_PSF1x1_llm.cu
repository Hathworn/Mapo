#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst) {
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    float cx = u_ * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v_ * time_scale + static_cast<float>(i) + 1.0f;

    // Precompute indices to minimize re-computation
    int tx = __float2int_rd(cx); // Floor rounding for more predictable position
    int ty = __float2int_rd(cy);

    // Fetch once to reduce repeated access
    float value = src[image_row_offset + j];

    // Check bounds and add atomically
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}