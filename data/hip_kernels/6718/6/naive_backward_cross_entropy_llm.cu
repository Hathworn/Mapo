#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naive_backward_cross_entropy(float *in, int *one_hot_classes, float inv_batches, int size, float *out)
{
    int bid = blockIdx.x * blockDim.x + threadIdx.x;
    if (bid < size) {
        // Simplified division by using pre-calculated reciprocal
        out[bid] = (in[bid] - one_hot_classes[bid]) * inv_batches;
    }
}