#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void warpReduce(volatile float *sdata, int tid) {
    // Unrolled loop for warp-level reduction
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}

__global__ void naive_sum(float *input, int size, float *out) {
    const unsigned int tid = threadIdx.x;
    const unsigned int bid = blockIdx.x * blockDim.x * 2 + tid;
    extern __shared__ float sdata[];

    if (!(bid < size)) return;

    // Load input into shared memory and do first level of reduction
    sdata[tid] = input[bid] + ((bid + blockDim.x < size) ? input[bid + blockDim.x] : 0);
    __syncthreads();

    // Reduce within the block using a loop
    for (unsigned int offset = blockDim.x / 2; offset > 32; offset /= 2) {
        if (tid < offset) {
            sdata[tid] += sdata[tid + offset];
        }
        __syncthreads();
    }

    // Perform warp-level reduction
    if (tid < 32) warpReduce(sdata, tid);

    // Write result for this block to global output
    if (tid == 0) out[blockIdx.x] = sdata[0];
}