#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void optimized_matrix_transpose(float *input, int axis_0, int axis_1, float *output)
{
    // Use shared memory with padding to avoid bank conflicts
    __shared__ float tile[TILE_DIM][TILE_DIM + 1];

    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
    
    // Load data into shared memory with coalesced access
    if (xIndex < axis_0) {
        for (int i = 0; i < TILE_DIM && (yIndex + i) < axis_1; i += BLOCK_HEIGHT) {
            tile[threadIdx.y + i][threadIdx.x] = input[(yIndex + i) * axis_0 + xIndex];
        }
    }

    __syncthreads();

    xIndex = blockIdx.y * TILE_DIM + threadIdx.x; // Transpose block offset
    yIndex = blockIdx.x * TILE_DIM + threadIdx.y;

    // Write transposed data from shared memory to global memory with coalesced access
    if (yIndex < axis_0) {
        for (int i = 0; i < TILE_DIM && (xIndex + i) < axis_1; i += BLOCK_HEIGHT) {
            output[(xIndex + i) * axis_0 + yIndex] = tile[threadIdx.x][threadIdx.y + i];
        }
    }
}