#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naive_bias_add(float *in, int size, float *bias, int bias_size)
{
    int bid = blockIdx.x * blockDim.x + threadIdx.x;
    if (bid < size) {
        // Compute bias offset using modulus instead of division/multiplication
        int bias_offset = bid % bias_size;
        in[bid] += bias[bias_offset];
    }
}