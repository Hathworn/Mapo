#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Calculate thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure that only one thread performs the division for accuracy
    if (idx == 0) {
        *accuracy /= N;
    }
}