#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pod_racing(unsigned int *d_rand, unsigned int *win, unsigned int *loss, unsigned int size, int *iter) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    __shared__ int shared_iter;  // Use shared memory for iter

    if (threadIdx.x == 0) {
        shared_iter = iter[0];
    }
    __syncthreads();

    const unsigned int flips[] = { 1, 1, 0, 1, 0, 1, 1, 1, 0, 1, 1, 1, 0, 1, 0, 1 };
    if (index < size) {
        // Check flip result and update win/loss accordingly
        if ((d_rand[index] % 2) != flips[shared_iter]) {
            shared_iter = 0;
            loss[index] = 1;
        } else {
            shared_iter++;
            if (shared_iter == 15) {
                win[index] = 1;
                shared_iter = 0;
            }
        }
    }

    __syncthreads();
    if (threadIdx.x == 0) {
        iter[0] = shared_iter;
    }
}