#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_arrays_gpu(unsigned int * a, unsigned int * b, int size)
{
    // Use shared memory for efficient data access
    __shared__ unsigned int shared_a;

    // Initialize shared memory only once
    if (threadIdx.x == 0)
        shared_a = a[0];
    
    __syncthreads();

    int index = blockDim.x * blockIdx.x + threadIdx.x;

    // Perform the computation and update shared memory
    if (index < size) {
        atomicAdd(&shared_a, b[index]);
    }

    __syncthreads();

    // Update global memory from shared memory
    if (threadIdx.x == 0)
        atomicAdd(&a[0], shared_a - a[0]);
}