#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel(int32_t n, float old_num_frames, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    float cur_weight = *cur_tot_weight;
    float new_num_frames = old_num_frames + cur_weight;
    float prior_scale_change = 1.0f;

    if (max_count != 0.0f) {
        float old_prior_scale = max(old_num_frames, (float)max_count) / max_count;
        float new_prior_scale = max(new_num_frames, (float)max_count) / max_count;
        prior_scale_change += new_prior_scale - old_prior_scale;
    }

    // Use shared memory to cache prior_scale_change for better performance
    __shared__ float shared_prior_scale_change;
    if (threadIdx.x == 0) {
        shared_prior_scale_change = prior_scale_change;
    }
    __syncthreads();

    for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        quadratic[diag_idx] += shared_prior_scale_change;
    }

    // Ensure only one thread updates linear
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        linear[0] += prior_offset * shared_prior_scale_change;
    }
}