#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float* log_energy, float* lifter_coeffs, float* features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    float* feats = features + frame * ldf;

    // Apply lifter coefficients with improved memory coalescing
    if (cepstral_lifter != 0.0f) {
        for (int c = thread_id; c < num_cols; c += blockDim.x) {
            feats[c] *= lifter_coeffs[c];
        }
    }

    // Use shared memory to possibly optimize energy access (hypothetical, usage depends on size and use case)
    __shared__ float shared_log_energy;
    
    // Apply energy in thread 0 for the frame only if needed
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = logf(energy_floor);  // Using logf for float computation

        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        
        shared_log_energy = energy;
        feats[0] = shared_log_energy;
    }
}