#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Calculate global index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds to avoid out of range memory access
    if (row < num_rows && col < num_cols) {
        float f = feats[row * ldf + col];
        feats_sq[row * lds + col] = f * f;
    }
}