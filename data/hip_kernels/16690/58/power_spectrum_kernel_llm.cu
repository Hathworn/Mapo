#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    const float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    
    // Using shared memory for performance improvement
    __shared__ float shared_A_in[CU1DBLOCK];
    __shared__ float shared_A_out[CU1DBLOCK];
    
    for (int idx = thread_id; idx < half_length; idx += CU1DBLOCK) {
        if (idx == 0) continue; // Avoid processing the special case in loop

        shared_A_in[thread_id] = reinterpret_cast<const float2 *>(Ar)[idx].x;
        shared_A_out[thread_id] = reinterpret_cast<const float2 *>(Ar)[idx].y;

        __syncthreads(); // Ensure all threads have loaded their data before proceeding
        
        float2 val = make_float2(shared_A_in[thread_id], shared_A_out[thread_id]);
        float ret = val.x * val.x + val.y * val.y;
        
        if (use_power) {
            Aw[idx] = ret;
        } else {
            Aw[idx] = sqrtf(ret);
        }
    }

    // Handle special case only once having 1 thread per block perform operation
    if (thread_id == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        if (use_power) {
            Aw[0] = real * real;
            Aw[half_length] = im * im;
        } else {
            Aw[0] = fabs(real);
            Aw[half_length] = fabs(im);
        }
    }
}