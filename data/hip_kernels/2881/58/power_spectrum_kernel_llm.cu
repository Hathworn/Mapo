#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for efficient looping
    const float *Ar = A_in + blockIdx.x * ldi;
    float *Aw = A_out + blockIdx.x * ldo;
    int half_length = row_length / 2;

    for (int idx = thread_id; idx < half_length; idx += stride) {
        if (idx == 0) continue; // Ignore special case
        float2 val = reinterpret_cast<const float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret); // Use ternary for concise power check
    }

    // Handle special case
    if (thread_id == 0) {
        float real = Ar[0];
        float im = Ar[row_length];
        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}