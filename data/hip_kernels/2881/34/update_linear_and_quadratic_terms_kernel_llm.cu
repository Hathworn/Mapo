#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel(int32_t n, float old_num_frames, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    float cur_weight = *cur_tot_weight;
    float new_num_frames = old_num_frames + cur_weight;
    float prior_scale_change = 1.0f;

    if (max_count != 0.0f) {
        float old_prior_scale = max(old_num_frames, (float)max_count) / max_count;
        float new_prior_scale = max(new_num_frames, (float)max_count) / max_count;
        prior_scale_change += new_prior_scale - old_prior_scale;
    }

    // Optimize loop by allowing each thread to handle multiple elements if necessary.
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride = blockDim.x * gridDim.x;

    for (; i < n; i += stride) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        quadratic[diag_idx] += prior_scale_change;
    }

    // Use shared memory to accumulate linear term from all threads in a block
    __shared__ float local_linear_update;

    if (threadIdx.x == 0) {
        local_linear_update = 0.0f;
    }
    __syncthreads();

    atomicAdd(&local_linear_update, (threadIdx.x == 0 && blockIdx.x == 0) ? prior_offset * prior_scale_change : 0.0f);
    __syncthreads();

    if (threadIdx.x == 0) {
        atomicAdd(&linear[0], local_linear_update);
    }
}