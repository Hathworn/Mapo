#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float generateRandom(hiprandState* globalState) {
    int ind = blockIdx.x * blockDim.x + threadIdx.x; // Ensure unique index for each thread
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform(&localState);
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void initializeBinary1Dpopulation(int *population, int sizeofPopulation, int sizeofChormosome, hiprandState* globalState, int division) {
    int populationIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (populationIndex < (sizeofPopulation * sizeofChormosome)) {
        population[populationIndex] = (int)(generateRandom(globalState) * 2);
        // No need for printf; it slows down execution
    }
}