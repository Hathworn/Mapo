#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float generateRandom( hiprandState* globalState)
{
//int ind = threadIdx.x;
hiprandState localState = globalState[0];
float RANDOM = hiprand_uniform( &localState );
globalState[0] = localState;
return RANDOM;
}
__global__ void initializeBinary1Dpopulation(int *population,int sizeofPopulation,int sizeofChormosome,hiprandState* globalState,int division){
int populationIndex =  blockIdx.x * blockDim.x + threadIdx.x;
if(populationIndex<(sizeofPopulation*sizeofChormosome)){
population[populationIndex]=(int) (generateRandom(globalState)*2);
//printf("CUDA %d\n",population[populationIndex]);

}
__syncthreads();
}