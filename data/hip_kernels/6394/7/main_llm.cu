#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "gpuCrossover_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    char *chromosome = NULL;
hipMalloc(&chromosome, XSIZE*YSIZE);
hiprandState *globalState = NULL;
hipMalloc(&globalState, XSIZE*YSIZE);
int sizeofChromosome = XSIZE*YSIZE;
int sizeofPopulation = XSIZE*YSIZE;
int Bias = 1;
float prob = 1;
char *range = NULL;
hipMalloc(&range, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
gpuCrossover<<<gridBlock, threadBlock>>>(chromosome,globalState,sizeofChromosome,sizeofPopulation,Bias,prob,range);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
gpuCrossover<<<gridBlock, threadBlock>>>(chromosome,globalState,sizeofChromosome,sizeofPopulation,Bias,prob,range);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
gpuCrossover<<<gridBlock, threadBlock>>>(chromosome,globalState,sizeofChromosome,sizeofPopulation,Bias,prob,range);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}