#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float generateRandom(hiprandState* globalState) {
    // Using thread index to ensure unique seeds for each thread
    int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform(&localState);
    globalState[ind] = localState;
    return RANDOM;
}

__device__ float generateRandomc(hiprandState* globalState) {
    // Using thread index to ensure unique seeds for each thread
    int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform(&localState);
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void initializeChar1Dpopulation(char *population, int sizeofPopulation, int sizeofChormosome, hiprandState* globalState, int division) {
    int populationIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (populationIndex < (sizeofPopulation * sizeofChormosome)) {
        // Directly generating random character without casting to int
        population[populationIndex] = (char)((generateRandomc(globalState) * 2) + 48);
    }
    // Removed __syncthreads() as it's not required based on current logic
}