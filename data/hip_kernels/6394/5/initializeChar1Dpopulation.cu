#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float generateRandom( hiprandState* globalState)
{
//int ind = threadIdx.x;
hiprandState localState = globalState[0];
float RANDOM = hiprand_uniform( &localState );
globalState[0] = localState;
return RANDOM;
}
__device__ float generateRandomc( hiprandState* globalState)
{
//int ind = threadIdx.x;
hiprandState localState = globalState[0];
float RANDOM = hiprand_uniform( &localState );
globalState[0] = localState;
return RANDOM;
}
__global__ void initializeChar1Dpopulation(char *population,int sizeofPopulation,int sizeofChormosome,hiprandState* globalState,int division){
int populationIndex =  blockIdx.x * blockDim.x + threadIdx.x;
if(populationIndex<(sizeofPopulation*sizeofChormosome)){
population[populationIndex]= (char) ((int) (generateRandomc(globalState)*2)+48);
//printf("CUDA %d\n",population[populationIndex]);

}
__syncthreads();
}