#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void Crossover(int *chromosome, int size, int start1, int end1, int start2, int end2, int mid) {
    for (int i = mid; i < size; i++) {
        int c1 = start1 + mid;
        int c2 = start2 + mid;
        int temp = chromosome[c1];
        chromosome[c1] = chromosome[c2];
        chromosome[c2] = temp;
    }
}

__device__ float generateRandom(hiprandState *globalState) {
    hiprandState localState = globalState[0];
    float RANDOM = hiprand_uniform(&localState);
    globalState[0] = localState;
    return RANDOM;
}

__device__ void Crossover(char *chromosome, int size, int start1, int end1, int start2, int end2, int mid) {
    for (int i = mid; i < size; i++) {
        int c1 = start1 + mid;
        int c2 = start2 + mid;
        int temp = chromosome[c1];
        chromosome[c1] = chromosome[c2];
        chromosome[c2] = temp;
    }
}

__device__ void Crossover(float *chromosome, int size, int start1, int end1, int start2, int end2, int mid) {
    for (int i = mid; i < size; i++) {
        int c1 = start1 + mid;
        int c2 = start2 + mid;
        float temp = chromosome[c1];
        chromosome[c1] = chromosome[c2];
        chromosome[c2] = temp;
    }
}

__global__ void gpuCrossover(int *chromosome, hiprandState *globalState, int sizeofChromosome, int sizeofPopulation, int Bias, float prob) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Generate crossover mid-point
    int mid = static_cast<int>(generateRandom(globalState) * sizeofChromosome);
    idx = idx * 2;
    int start1 = idx * sizeofChromosome;
    int end1 = start1 + sizeofChromosome;
    int start2 = end1;
    int end2 = start2 + sizeofChromosome;

    // Ensure indices are within limits
    if (end2 < (sizeofChromosome * sizeofPopulation))
        Crossover(chromosome, sizeofChromosome, start1, end1, start2, end2, mid);

    // Perform mutation with probability
    if ((int)(generateRandom(globalState) * 100) < (prob * 100)) {
        int j = static_cast<int>(generateRandom(globalState) * (sizeofChromosome / 4));
        for (int k = 0; k < j; k++) {
            int index = static_cast<int>(generateRandom(globalState) * sizeofChromosome);
            chromosome[index] = 1 - chromosome[index];  // Toggle between 0 and 1
        }
    }
}