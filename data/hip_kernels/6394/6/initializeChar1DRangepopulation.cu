#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float generateRandom( hiprandState* globalState)
{
//int ind = threadIdx.x;
hiprandState localState = globalState[0];
float RANDOM = hiprand_uniform( &localState );
globalState[0] = localState;
return RANDOM;
}
__device__ float generateRandomc( hiprandState* globalState)
{
//int ind = threadIdx.x;
hiprandState localState = globalState[0];
float RANDOM = hiprand_uniform( &localState );
globalState[0] = localState;
return RANDOM;
}
__global__ void initializeChar1DRangepopulation(char *population,int sizeofPopulation,int sizeofChormosome,hiprandState* globalState,int division,char* range){
int populationIndex =  blockIdx.x * blockDim.x + threadIdx.x;
if(populationIndex<(sizeofPopulation*sizeofChormosome)){
population[populationIndex]= range[(int) (generateRandomc(globalState)*sizeofChormosome)];
//printf("CUDA %d\n",population[populationIndex]);

}
__syncthreads();
}