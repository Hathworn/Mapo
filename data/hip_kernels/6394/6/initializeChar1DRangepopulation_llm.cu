#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float generateRandom(hiprandState* globalState) {
    hiprandState localState = globalState[0];
    float RANDOM = hiprand_uniform(&localState);
    globalState[0] = localState;
    return RANDOM;
}

__device__ float generateRandomc(hiprandState* globalState) {
    hiprandState localState = globalState[0];
    float RANDOM = hiprand_uniform(&localState);
    globalState[0] = localState;
    return RANDOM;
}

__global__ void initializeChar1DRangepopulation(char *population, int sizeofPopulation, int sizeofChormosome, hiprandState* globalState, int division, char* range) {
    int populationIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread can process a population index
    if(populationIndex < (sizeofPopulation * sizeofChormosome)) {
        // Use shared memory for faster access if multiple threads use the same data
        __shared__ hiprandState sharedState;
        if (threadIdx.x == 0) {
            sharedState = globalState[0];
        }
        __syncthreads();

        // Generate random number and store result in local population array
        float randValue = hiprand_uniform(&sharedState);
        if (threadIdx.x == 0) {
            globalState[0] = sharedState;
        }
        __syncthreads();

        population[populationIndex] = range[(int)(randValue * sizeofChormosome)];
    }
}