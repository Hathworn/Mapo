#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h" // Ensure hiprand header is included

using namespace std;

float *valuesf;
float *weightf;
float maxWf;

float *matchf;
const int fSUMFLAG=0;
const int fKNAPSACKFLAG = 1;

const int fAVGFLAG=2;
const int fMATCHFLAG=3;
const int fINVERSESUMFLAG=4;

const int fMAXIMIZE=-1;
const int fMINIMIZE=1;

__device__ void Crossover(int *chromosome, int size, int start1, int end1, int start2, int end2, int mid) {
    for (int i = mid; i < size; i++) {
        int c1 = start1 + mid;
        int c2 = start2 + mid;
        int temp = chromosome[c1];
        
        chromosome[c1] = chromosome[c2];
        chromosome[c2] = temp;
    }
}

__device__ float generateRandom(hiprandState* globalState) {
    hiprandState localState = globalState[0];
    float RANDOM = hiprand_uniform(&localState);
    globalState[0] = localState;
    return RANDOM;
}

__device__ void Crossover(char *chromosome, int size, int start1, int end1, int start2, int end2, int mid) {
    for (int i = mid; i < size; i++) {
        int c1 = start1 + mid;
        int c2 = start2 + mid;
        int temp = chromosome[c1];
        
        chromosome[c1] = chromosome[c2];
        chromosome[c2] = temp;
    }
}

__device__ void Crossover(float *chromosome, int size, int start1, int end1, int start2, int end2, int mid) {
    for (int i = mid; i < size; i++) {
        int c1 = start1 + mid;
        int c2 = start2 + mid;
        float temp = chromosome[c1]; // Use correct type for temp
        
        chromosome[c1] = chromosome[c2];
        chromosome[c2] = temp;
    }
}

__device__ float generateRandomf(hiprandState* globalState) {
    hiprandState localState = globalState[0];
    float RANDOM = hiprand_uniform(&localState);
    globalState[0] = localState;
    return RANDOM;
}

__global__ void gpuCrossover(float *chromosome, hiprandState *globalState, int sizeofChromosome, int sizeofPopulation, int Bias, float prob) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int mid = static_cast<int>(generateRandomf(globalState) * sizeofChromosome);
    idx = idx * 2;
    int start1, end1;

    int start2, end2;
    start1 = idx * sizeofChromosome;
    end1 = start1 + sizeofChromosome;
    start2 = end1;
    end2 = start2 + sizeofChromosome;
    
    // Ensure the crossover happens within bounds
    if (end2 < (sizeofChromosome * sizeofPopulation))
        Crossover(chromosome, sizeofChromosome, start1, end1, start2, end2, mid);

    int number = static_cast<int>(generateRandomf(globalState) * 100);
    if (number < (prob * 100)) {
        int j = static_cast<int>(generateRandomf(globalState) * (sizeofChromosome / 4));
        for (int k = 0; k < j; k++) {
            int index = static_cast<int>(generateRandomf(globalState) * sizeofChromosome);
            float a = chromosome[index];
            chromosome[index] = (a == 1.0f) ? 0.0f : 1.0f;
        }
    }
}