#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_kernelc ( hiprandState *state, unsigned long seed )
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate unique index for each thread
    hiprand_init(seed, idx, 0, &state[idx]); // Initialize RNG state for each thread
}