#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState *state, unsigned long seed)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate unique thread ID
    hiprand_init(seed, id, 0, &state[id]);            // Initialize state with unique ID
}