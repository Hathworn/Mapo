#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_kernel ( hiprandState *state, unsigned long seed )
{
hiprand_init ( seed, 0, 0, &state[0] );
}