#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

float *valuesf;
float *weightf;
float maxWf;

float *matchf;
const int fSUMFLAG=0;
const int fKNAPSACKFLAG = 1;

const int fAVGFLAG=2;
const int fMATCHFLAG=3;
const int fINVERSESUMFLAG=4;

const int fMAXIMIZE=-1;
const int fMINIMIZE=1;

__global__ void setup_kernelf(hiprandState *state, unsigned long seed)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x; // Compute unique thread ID
    hiprand_init(seed, id, 0, &state[id]); // Initialize hiprand state for each thread
}