#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"



using namespace std;


float *valuesf;
float *weightf;
float maxWf;

float *matchf;
const int fSUMFLAG=0;
const int fKNAPSACKFLAG = 1;

const int fAVGFLAG=2;
const int fMATCHFLAG=3;
const int fINVERSESUMFLAG=4;

const int fMAXIMIZE=-1;
const int fMINIMIZE=1;




__device__ float generateRandom( hiprandState* globalState)
{
//int ind = threadIdx.x;
hiprandState localState = globalState[0];
float RANDOM = hiprand_uniform( &localState );
globalState[0] = localState;
return RANDOM;
}
__device__ float generateRandomf( hiprandState* globalState)
{
//int ind = threadIdx.x;
hiprandState localState = globalState[0];
float RANDOM = hiprand_uniform( &localState );
globalState[0] = localState;
return RANDOM;
}
__global__ void initializeFloat1Dpopulation(float *population,int sizeofPopulation,int sizeofChormosome,hiprandState* globalState,int division){
int populationIndex =  blockIdx.x * blockDim.x + threadIdx.x;
if(populationIndex<(sizeofPopulation*sizeofChormosome)){
population[populationIndex]= (int)(generateRandomf(globalState)*2);
//printf("CUDA %d\n",population[populationIndex]);

}
__syncthreads();
}