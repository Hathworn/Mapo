#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

float *valuesf;
float *weightf;
float maxWf;

float *matchf;
const int fSUMFLAG=0;
const int fKNAPSACKFLAG = 1;

const int fAVGFLAG=2;
const int fMATCHFLAG=3;
const int fINVERSESUMFLAG=4;

const int fMAXIMIZE=-1;
const int fMINIMIZE=1;

__device__ float generateRandom(hiprandState* globalState) {
    hiprandState localState = globalState[0];
    float RANDOM = hiprand_uniform(&localState);
    globalState[0] = localState;
    return RANDOM;
}

__device__ float generateRandomf(hiprandState* globalState) {
    hiprandState localState = globalState[0];
    float RANDOM = hiprand_uniform(&localState);
    globalState[0] = localState;
    return RANDOM;
}

__global__ void initializeFloat1Dpopulation(float *population, int sizeofPopulation, int sizeofChormosome, hiprandState* globalState, int division) {
    int populationIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (populationIndex < (sizeofPopulation * sizeofChormosome)) {
        // Generate binary value in a more concise way
        population[populationIndex] = generateRandomf(globalState) < 0.5f ? 0 : 1;
    }
    // __syncthreads() is not needed when each thread operates on independent memory
}