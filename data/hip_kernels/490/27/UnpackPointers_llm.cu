#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    const int blockStart = tx + blockIdx.x * blockDim.x;
    const int stride = blockDim.x * gridDim.x; // Calculate stride for processing

    int numPts = 0;

    // Loop with stride for better utilization of threads
    for (int pos = blockStart; pos < h / 32 * w; pos += stride) {
        int y = pos / w;
        int x = pos % w;

        if (x + tx < w) {
            unsigned int val = minmax[y * w + x + tx];
            if (val) {
                for (int k = 0; k < 32 && numPts < maxPts; k++) {
                    if (val & 0x1) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}