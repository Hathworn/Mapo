#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    const int b = blockDim.x;
    int p = y * width + x;

    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
        for (int ty = 1; ty < 16; ++ty) {
            p += width;
            val = d_Data[p];
            if (val < minvals[tx]) minvals[tx] = val; // Update minimum
            if (val > maxvals[tx]) maxvals[tx] = val; // Update maximum
        }
    } else {
        minvals[tx] = FLT_MAX;
        maxvals[tx] = -FLT_MAX;
    }

    __syncthreads();

    // Perform reduction within the block to find global min/max
    for (int d = b / 2; d > 0; d >>= 1) {
        if (tx < d) {
            float tempMin = minvals[tx + d];
            float tempMax = maxvals[tx + d];
            if (tempMin < minvals[tx]) minvals[tx] = tempMin; // Reduce min
            if (tempMax > maxvals[tx]) maxvals[tx] = tempMax; // Reduce max
        }
        __syncthreads();
    }

    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}