#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Compute the global thread coordinates
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;
    
    // Check if within bounds and perform subtraction
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // No need for __syncthreads() here as there are no shared memory accesses
}