#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;
    // Use y_step to avoid division inside the loop
    for (int y = 0, y_step = 0; y < h; y += 32, y_step += w) {
        for (int x = 0; x < w; x += 16) {
            unsigned int val = minmax[y_step + x + tx];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = (y + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}