#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int pitch, int height)
{
    // Using shared memory for data caching.
    __shared__ float s_data1[3 * (MINMAX_W + 2)];
    __shared__ float s_data2[3 * (MINMAX_W + 2)];
    __shared__ float s_data3[3 * (MINMAX_W + 2)];

    // Shared memory for min and max calculations within a tile.
    __shared__ float s_ymin1[(MINMAX_W + 2)], s_ymin2[(MINMAX_W + 2)], s_ymin3[(MINMAX_W + 2)];
    __shared__ float s_ymax1[(MINMAX_W + 2)], s_ymax2[(MINMAX_W + 2)], s_ymax3[(MINMAX_W + 2)];

    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_W;
    const int xEndClamped = min(xStart + MINMAX_W - 1, width - 1);
    const int xReadPos = xStart + tx - WARP_SIZE;
    const int xWritePos = xStart + tx;
    int memWid = MINMAX_W + 2;

    int memPos0 = (tx - WARP_SIZE + 1);
    int memPos1 = memPos0;
    int yIndex = 0;
    unsigned int output = 0;

    for (int y = 0; y < 34; ++y) { // Merged loop init with reset for y.
        output >>= 1; // Right shift output bits.

        int memPos = yIndex * memWid + memPos0;
        int yp = 32 * blockIdx.y + y - 1;
        yp = min(max(yp, 0), height - 1);
        int readStart = yp * pitch;

        // Load data into shared memory.
        if (tx >= (WARP_SIZE - 1)) {
            float fetch1 = (xReadPos >= 0 && xReadPos < width) ? d_Data1[readStart + xReadPos] : 0;
            float fetch2 = (xReadPos >= 0 && xReadPos < width) ? d_Data2[readStart + xReadPos] : 0;
            float fetch3 = (xReadPos >= 0 && xReadPos < width) ? d_Data3[readStart + xReadPos] : 0;
            s_data1[memPos] = fetch1;
            s_data2[memPos] = fetch2;
            s_data3[memPos] = fetch3;
        }
        __syncthreads();

        int memPos2 = yIndex * memWid + tx;
        if (y > 1) { // Min-max logic applied after two initial rows are read.
            if (tx < memWid) {
                float min1 = fminf(fminf(s_data1[memPos0], s_data1[memPos1]), s_data1[memPos2]);
                float min2 = fminf(fminf(s_data2[memPos0], s_data2[memPos1]), s_data2[memPos2]);
                float min3 = fminf(fminf(s_data3[memPos0], s_data3[memPos1]), s_data3[memPos2]);
                float max1 = fmaxf(fmaxf(s_data1[memPos0], s_data1[memPos1]), s_data1[memPos2]);
                float max2 = fmaxf(fmaxf(s_data2[memPos0], s_data2[memPos1]), s_data2[memPos2]);
                float max3 = fmaxf(fmaxf(s_data3[memPos0], s_data3[memPos1]), s_data3[memPos2]);

                s_ymin1[tx] = min1;
                s_ymin2[tx] = fminf(fminf(min1, min2), min3);
                s_ymin3[tx] = min3;
                s_ymax1[tx] = max1;
                s_ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
                s_ymax3[tx] = max3;
            }
        }
        __syncthreads();

        if (y > 1 && tx < MINMAX_W && xWritePos <= xEndClamped) { // Range checks prior to writing output.
            float minv = fminf(fminf(fminf(fminf(fminf(s_ymin2[tx], s_ymin2[tx + 2]), s_ymin1[tx + 1]), s_ymin3[tx + 1]), s_data2[memPos0 + 1]), s_data2[memPos2 + 1]);
            minv = fminf(minv, d_Threshold[1]);
            float maxv = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(s_ymax2[tx], s_ymax2[tx + 2]), s_ymax1[tx + 1]), s_ymax3[tx + 1]), s_data2[memPos0 + 1]), s_data2[memPos2 + 1]);
            maxv = fmaxf(maxv, d_Threshold[0]);

            if (s_data2[memPos1 + 1] < minv || s_data2[memPos1 + 1] > maxv)
                output |= 0x80000000; // Set flag for result.
        }
        __syncthreads();

        memPos0 = memPos1;
        memPos1 = memPos2;
        yIndex = (yIndex < 2) ? yIndex + 1 : 0; // Cycles through memory positions for caching.
    }

    if (tx < MINMAX_W && xWritePos < width) {
        int writeStart = blockIdx.y * pitch + xWritePos;
        d_Result[writeStart] = output;
    }
}