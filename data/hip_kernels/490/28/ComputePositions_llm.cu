#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputePositions(float *g_Data1, float *g_Data2, float *g_Data3, int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
    int i = blockIdx.x * POSBLK_SIZE + threadIdx.x;
    if (i >= numPts)
        return;

    int p = d_Ptrs[i];

    // Precompute indices
    int pw = p - w;
    int pw1 = p - 1;
    int pw_1 = pw - 1;
    int p1 = p + 1;

    float val[7];
    val[0] = g_Data2[p];
    val[1] = g_Data2[pw1];
    val[2] = g_Data2[p1];

    // Optimize computations
    float dx = 0.5f * (val[2] - val[1]);
    float dxx = 2.0f * val[0] - val[1] - val[2];
    
    val[3] = g_Data2[pw];
    val[4] = g_Data2[p + w];

    float dy = 0.5f * (val[4] - val[3]);
    float dyy = 2.0f * val[0] - val[3] - val[4];
    
    val[5] = g_Data3[p];
    val[6] = g_Data1[p];

    float ds = 0.5f * (val[6] - val[5]);
    float dss = 2.0f * val[0] - val[5] - val[6];

    int wp1 = p + w + 1;
    float dxy = 0.25f * (g_Data2[wp1] + g_Data2[pw_1] - g_Data2[p - w + 1] - g_Data2[p + w - 1]);

    float dxs = 0.25f * (g_Data3[p1] + g_Data1[pw1] - g_Data1[p1] - g_Data3[pw1]);
    float dys = 0.25f * (g_Data3[p + w] + g_Data1[pw] - g_Data3[pw] - g_Data1[p + w]);

    // Precompute inverse matrix
    float idxx = dyy * dss - dys * dys;
    float idxy = dys * dxs - dxy * dss;
    float idxs = dxy * dys - dyy * dxs;
    float idyy = dxx * dss - dxs * dxs;
    float idys = dxy * dxs - dxx * dys;
    float idss = dxx * dyy - dxy * dxy;

    float det = idxx * dxx + idxy * dxy + idxs * dxs;
    float idet = 1.0f / det;

    float pdx = idet * (idxx * dx + idxy * dy + idxs * ds);
    float pdy = idet * (idxy * dx + idyy * dy + idys * ds);
    float pds = idet * (idxs * dx + idys * dy + idss * ds);

    // Correct displacement if excessive
    if (pdx < -0.5f || pdx > 0.5f || pdy < -0.5f || pdy > 0.5f || pds < -0.5f || pds > 0.5f) {
        pdx = __fdividef(dx, dxx);
        pdy = __fdividef(dy, dyy);
        pds = __fdividef(ds, dss);
    }

    float dval = 0.5f * (dx * pdx + dy * pdy + ds * pds);
    d_Sift[i] = (p % w) + pdx;
    d_Sift[i + maxPts] = (p / w) + pdy;
    d_Sift[i + 2 * maxPts] = d_ConstantA[0] * exp2f(pds * d_ConstantB[0]);
    d_Sift[i + 3 * maxPts] = val[0] + dval;

    // Calculate and store trace and determinant
    float tra = dxx + dyy;
    det = dxx * dyy - dxy * dxy;
    d_Sift[i + 4 * maxPts] = __fdividef(tra * tra, det);
}