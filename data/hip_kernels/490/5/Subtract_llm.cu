#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    // Calculate the global index once outside the if statement to avoid recomputation
    const int x = blockIdx.x * SUBTRACT_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACT_H + threadIdx.y;

    // Check bounds before accessing memory
    if (x < width && y < height) {
        int p = y * pitch + x;
        // Perform the subtraction and store the result
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // Remove unnecessary __syncthreads()
}