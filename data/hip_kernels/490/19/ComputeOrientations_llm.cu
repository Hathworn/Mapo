#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];

    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Initialize histogram bins
    for (int i = 0; i < 13; i++)
        hist[i * 32 + tx] = 0.0f;
    __syncthreads();

    // Precompute Gaussian values
    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    if (tx < 15)
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));

    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;
    int px = xp & 15;
    int x = tx - px;

    // Load data into shared memory
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = xp + x;
        int yi = yp + y;
        if (xi < 0) xi = 0;
        if (xi >= w) xi = w - 1;
        if (yi < 0) yi = 0;
        if (yi >= h) yi = h - 1;
        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    // Compute gradients and fill histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = 16.0f * atan2f(dy, dx) / 3.1416f + 16.5f;
            if (bin == 32)
                bin = 0;
            float grad = sqrtf(dx * dx + dy * dy);
            atomicAdd(&hist[32 * (x - 1) + bin], grad * gauss[x] * gauss[y]);
        }
    }
    __syncthreads();

    // Histogram reduction
    for (int stride = 8; stride > 0; stride /= 2) {
        if (y * 2 + stride < 13 * 32)
            hist[y * 32 + tx] += hist[(y + stride) * 32 + tx];
        __syncthreads();
    }

    // Smooth histogram
    if (tx < 32) {
        float v = 6 * hist[tx] + 4 * (hist[(tx + 1) % 32] + hist[(tx + 31) % 32]) + (hist[(tx + 2) % 32] + hist[(tx + 30) % 32]);
        hist[32 + tx] = v;
    }
    __syncthreads();

    // Find peaks in the histogram
    if (tx == 0) {
        float maxval1 = 0.0f, maxval2 = 0.0f;
        int i1 = -1, i2 = -1;
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }
        // Calculate and store orientations
        d_Orient[bx] = calculateOrientation(hist, i1, maxval1);
        d_Orient[bx + maxPts] = (maxval2 >= 0.8f * maxval1) ? calculateOrientation(hist, i2, maxval2) : -1;
    }
}

// Helper function to compute orientation
__device__ float calculateOrientation(float *hist, int i, float maxval) {
    float val1 = hist[32 + ((i + 1) % 32)];
    float val2 = hist[32 + ((i + 31) % 32)];
    float peak = i + 0.5f * (val1 - val2) / (2.0f * maxval - val1 - val2);
    return 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
}