#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LowPassRowMulti(float *d_Result, float *d_Data, int width, int pitch, int height) {
    __shared__ float data[CONVROW_W + 2*RADIUS];
    const int tx = threadIdx.x;
    const int block = blockIdx.x / (NUM_SCALES + 3);
    const int scale = blockIdx.x - (NUM_SCALES + 3) * block;
    const int xout = block * CONVROW_W + tx;
    const int loadPos = xout - RADIUS;
    const int yptr = blockIdx.y * pitch;
    const int writePos = yptr + height * pitch * scale + xout;
    float *kernel = d_Kernel + scale * 16;

    // Load data with boundary checks using min/max to avoid if-else branching
    data[tx] = d_Data[yptr + max(0, min(width - 1, loadPos))];
    __syncthreads();

    // Ensure threads within bounds to avoid unnecessary computation
    if (xout < width && tx < CONVROW_W) {
        d_Result[writePos] =
            (data[tx] + data[tx + 8]) * kernel[0] +
            (data[tx + 1] + data[tx + 7]) * kernel[1] +
            (data[tx + 2] + data[tx + 6]) * kernel[2] +
            (data[tx + 3] + data[tx + 5]) * kernel[3] +
            data[tx + 4] * kernel[4];
    }
}