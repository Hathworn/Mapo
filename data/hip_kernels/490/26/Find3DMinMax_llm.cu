#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel function
__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int height)
{
    // Improved shared memory usage with minimal allocation
    __shared__ float data1[MINMAX_SIZE + 2];
    __shared__ float data2[MINMAX_SIZE + 2];
    __shared__ float data3[MINMAX_SIZE + 2];
    __shared__ float ymin1[MINMAX_SIZE];
    __shared__ float ymin2[MINMAX_SIZE];
    __shared__ float ymin3[MINMAX_SIZE];
    __shared__ float ymax1[MINMAX_SIZE];
    __shared__ float ymax2[MINMAX_SIZE];
    __shared__ float ymax3[MINMAX_SIZE];

    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_SIZE;
    const int xEndClamped = min(xStart + MINMAX_SIZE - 1, width - 1);
    const int xReadPos = xStart + tx - WARP_SIZE + 1;
    const int xWritePos = xStart + tx;
    int yq = 0;
    unsigned int output = 0;

    // Read data within a vertical strip
    for (int y = 0; y < 34; y++) {
        output >>= 1;
        int yp = max(32 * blockIdx.y + y - 1, 0);
        yp = min(yp, height - 1);
        int readStart = yp * width;

        // Cache data when within bounds
        if (tx >= WARP_SIZE - 1) {
            if (xReadPos >= 0 && xReadPos < width) {
                data1[tx] = d_Data1[readStart + xReadPos];
                data2[tx] = d_Data2[readStart + xReadPos];
                data3[tx] = d_Data3[readStart + xReadPos];
            } else {
                data1[tx] = 0;
                data2[tx] = 0;
                data3[tx] = 0;
            }
        }
        __syncthreads();

        int memPos2 = yq * (MINMAX_SIZE + 2) + tx;
        if (y > 1 && tx < MINMAX_SIZE + 2) {
            float min1 = fminf(fminf(data1[tx - 1], data1[tx]), data1[tx + 1]);
            float min2 = fminf(fminf(data2[tx - 1], data2[tx]), data2[tx + 1]);
            float min3 = fminf(fminf(data3[tx - 1], data3[tx]), data3[tx + 1]);
            float max1 = fmaxf(fmaxf(data1[tx - 1], data1[tx]), data1[tx + 1]);
            float max2 = fmaxf(fmaxf(data2[tx - 1], data2[tx]), data2[tx + 1]);
            float max3 = fmaxf(fmaxf(data3[tx - 1], data3[tx]), data3[tx + 1]);
            ymin1[tx] = min1;
            ymin2[tx] = fminf(fminf(min1, min2), min3);
            ymin3[tx] = min3;
            ymax1[tx] = max1;
            ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
            ymax3[tx] = max3;
        }
        __syncthreads();

        if (y > 1 && tx < MINMAX_SIZE && xWritePos <= xEndClamped) {
            float minv = fminf(fminf(fminf(fminf(fminf(ymin2[tx], ymin2[tx + 2]), ymin1[tx + 1]), ymin3[tx + 1]), data2[tx]), d_ConstantA[1]);
            float maxv = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(ymax2[tx], ymax2[tx + 2]), ymax1[tx + 1]), ymax3[tx + 1]), data2[tx]), d_ConstantA[0]);
            if (data2[tx] < minv || data2[tx] > maxv)
                output |= 0x80000000;
        }
        __syncthreads();

        yq = (yq < 2 ? yq + 1 : 0);
    }

    // Write results back to global memory
    if (tx < MINMAX_SIZE && xWritePos < width) {
        d_Result[blockIdx.y * width + xWritePos] = output;
    }
}