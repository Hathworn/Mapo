#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Calculate the global thread indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Compute linear index in the data array
    int p = y * width + x;
    
    // Ensure within bounds access
    if (x < width && y < height)
    {
        // Perform the multiply-add operation
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
}
```
