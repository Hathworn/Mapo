#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * blockDim.x + tx; // Use blockDim.x for better flexibility
    const int y = blockIdx.y * blockDim.y;

    float minval = FLT_MAX;
    float maxval = -FLT_MAX;

    // Load data and find local min/max
    if (x < width) {
        for (int ty = 0; ty < 16 && (y + ty) < height; ty++) {
            int p = (y + ty) * pitch + x;
            float val = d_Data[p];
            minval = fminf(minval, val);
            maxval = fmaxf(maxval, val);
        }
    }

    minvals[tx] = minval;
    maxvals[tx] = maxval;
    __syncthreads();

    // Reduce within block
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (tx < offset) {
            minvals[tx] = fminf(minvals[tx], minvals[tx + offset]);
            maxvals[tx] = fmaxf(maxvals[tx], maxvals[tx + offset]);
        }
        __syncthreads();
    }

    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}