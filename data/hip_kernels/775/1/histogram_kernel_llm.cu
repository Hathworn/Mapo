#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogram_kernel(int* PartialHist, int* DeviceData, int DataCount, int* timer)
{
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    clock_t start_atomic = 0;
    clock_t stop_atomic = 0;

    extern __shared__ int hist[];

    // Initialize shared memory histogram to zero
    for (int i = tid; i < H * blockDim.x; i += blockDim.x) {
        hist[i] = 0;
    }

    __syncthreads();

    // Populate shared memory histogram
    for (int j = gid; j < DataCount; j += stride) {
        atomicAdd(&hist[tid * H + DeviceData[j]], 1);
    }

    __syncthreads();

    // Aggregate results into global memory
    for (int i = tid; i < H; i += blockDim.x) {
        int sum = 0;
        for (int t_hist = 0; t_hist < blockDim.x; t_hist++) {
            sum += hist[t_hist * H + i];
        }
        atomicAdd(&PartialHist[i], sum);
    }

    if (tid == 0) {
        start_atomic = clock();
    }

    __syncthreads();

    if (tid == 0) {
        stop_atomic = clock();
        timer[blockIdx.x] = stop_atomic - start_atomic;
    }
}