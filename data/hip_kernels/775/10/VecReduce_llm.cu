#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecReduce(float* g_idata, float* g_odata, int N)
{
    // shared memory size declared at kernel launch
    extern __shared__ float sdata[];

    // Load thread and global indices
    unsigned int tid = threadIdx.x;
    unsigned int globalid = blockIdx.x * blockDim.x + tid;

    // For thread ids greater than data space
    if (globalid < N) {
        sdata[tid] = g_idata[globalid];
    } else {
        sdata[tid] = 0;  // Case of extra threads above N
    }

    // Each thread loads one element from global to shared mem
    __syncthreads();

    // Do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global mem
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}