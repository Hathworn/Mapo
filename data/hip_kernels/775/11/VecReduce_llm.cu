#include "hip/hip_runtime.h"

```cpp
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecReduce(float* g_idata, float* g_odata, int N)
{
    extern __shared__ float sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int globalid = blockIdx.x * blockDim.x + threadIdx.x;

    // Load valid elements into shared memory, else zero
    sdata[tid] = (globalid < N) ? g_idata[globalid] : 0;
    __syncthreads();

    // Reduce using shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s]; // Perform addition directly
        }
        __syncthreads();
    }

    // Write result to global memory using atomic addition
    if (tid == 0) {
        atomicAdd(g_odata, sdata[0]);
    }
}