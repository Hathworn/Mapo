#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define H 	64

// Default values
int N = 10000; 		//Size
int T = 32; 		//BlockSize
int B = 4; 		//Blocks

// Host Variables
int* HostData;
int* HostHist;
int* HostTimer=NULL;

// Device Variables
int* DeviceData;
int* DeviceHist;
int* DeviceTimer=NULL;

// Timer Variables
struct timeval CPU_Time_start, CPU_Time_end;
struct timeval GPU_Time_start, GPU_Time_end;
struct timeval DeviceToHost_start, DeviceToHost_end;
struct timeval HostToDevice_start, HostToDevice_end;
struct timeval CPU_Partial_Time_start, CPU_Partial_Time_end;
struct timeval CPU_Cleanup_Time_start, CPU_Cleanup_Time_end;
struct timeval Total_Time_start, Total_Time_end;


// Function Declaration
void Cleanup(void);
void HistogramSequential(int* result, int* data, int size);

// Histogram kernel

__global__ void histogram_kernel(int* PartialHist, int* DeviceData, int dataCount, int* timer)
{
    // Get thread ID and global ID
    unsigned int tid = threadIdx.x;
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    clock_t start_clock = 0;
    clock_t stop_clock = 0;

    // Start the timer in the first thread of the block
    if (tid == 0)
    {
        start_clock = clock();
    }

    // Shared memory for block histogram
    __shared__ int BlockHist[H];
    __shared__ int hist[H];

    // Initialize shared memory
    BlockHist[tid] = 0;
    if (tid < H)
    {
        hist[tid] = 0;
    }
    __syncthreads();

    // Populate shared memory hist
    for (int pos = gid; pos < dataCount; pos += stride)
    {
        atomicAdd(&hist[DeviceData[pos]], 1);
    }
    __syncthreads();

    // Aggregate hist into BlockHist
    if (tid < H)
    {
        BlockHist[tid] = hist[tid];
    }
    __syncthreads();

    // Write to global memory
    if (tid < H)
    {
        PartialHist[blockIdx.x * H + tid] = BlockHist[tid];
    }
    
    // Stop the timer in the first thread of the block
    if (tid == 0)
    {
        stop_clock = clock();
        timer[blockIdx.x * 2] = start_clock;
        timer[blockIdx.x * 2 + 1] = stop_clock;
    }
}