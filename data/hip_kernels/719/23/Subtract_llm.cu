#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Simplified block and thread indexing
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int p = y * width + x;
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // Removed __syncthreads() as it's unnecessary in the current context
}