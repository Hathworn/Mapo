#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    // Calculate global thread index
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within bounds
    if (x < width && y < height) {
        int p = y * pitch + x;
        // Perform subtraction
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}