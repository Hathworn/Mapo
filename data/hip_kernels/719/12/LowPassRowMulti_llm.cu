#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LowPassRowMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    __shared__ float data[CONVROW_W + 2*RADIUS];

    const int tx = threadIdx.x;
    const int block = blockIdx.x / (NUM_SCALES + 3);
    const int scale = blockIdx.x % (NUM_SCALES + 3);
    const int xout = block * CONVROW_W + tx;
    const int loadPos = xout - RADIUS;
    const int yptr = blockIdx.y * pitch;
    const int writePos = yptr + height * pitch * scale + xout;

    float *kernel = d_Kernel + scale * 16;

    // Load data into shared memory with boundary checks
    if (loadPos < 0)
        data[tx] = d_Data[yptr];
    else if (loadPos >= width)
        data[tx] = d_Data[yptr + width - 1];
    else
        data[tx] = d_Data[yptr + loadPos];

    __syncthreads();

    // Perform convolution only if xout is within the image width
    if (xout < width && tx < CONVROW_W) {
        float sum = 0.0f;
        #pragma unroll // Unroll loop for better performance
        for (int i = 0; i < 5; ++i) {
            sum += (data[tx + i] + data[tx + 8 - i]) * kernel[i];
        }
        sum += data[tx+4] * kernel[4];
        d_Result[writePos] = sum;
    }
}