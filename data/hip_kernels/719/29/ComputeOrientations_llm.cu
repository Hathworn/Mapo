#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32];
    __shared__ float gauss[16];
    
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Initialize shared memory histogram to zero
    hist[tx] = 0.0f;
    __syncthreads();

    // Calculate Gaussian values for weighting
    float i2sigma2 = -1.0f/(2.0f*3.0f*3.0f);
    if (tx < 15)
        gauss[tx] = expf(i2sigma2 * (tx-7) * (tx-7));

    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;
    int px = xp & 15;
    int x = tx - px;

    // Load data with boundary check
    for (int y = 0; y < 15; y++) {
        int memPos = 16*y + x;
        int xi = min(max(xp + x, 0), w-1);
        int yi = min(max(yp + y, 0), h-1);
        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    // Compute gradients and binning
    for (int y = 1; y < 14; y++) {
        int memPos = 16*y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos+16] - data[memPos-16];
            float dx = data[memPos+1] - data[memPos-1];
            int bin = (int)(16.0f * atan2f(dy, dx) / 3.1416f + 16.5f) & 31;  // Use bitwise AND for modulo operation
            float grad = sqrtf(dx*dx + dy*dy);
            atomicAdd(&hist[bin], grad * gauss[x] * gauss[y]);  // Use atomic addition to avoid race conditions
        }
    }
    __syncthreads();

    // Compute peaks and orientations
    if (tx == 0) {
        // Find top two peaks
        float maxval1 = 0.0f, maxval2 = 0.0f;
        int i1 = -1, i2 = -1;
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }
        // Compute orientation based on peak
        float val1 = hist[(i1+1)&31];
        float val2 = hist[(i1+31)&31];
        float peak = i1 + 0.5f * (val1-val2) / (2.0f*maxval1 - val1 - val2);
        d_Orient[bx] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);

        // Handle second peak if significant
        if (maxval2 >= 0.8f * maxval1) {
            val1 = hist[(i2+1)&31];
            val2 = hist[(i2+31)&31];
            peak = i2 + 0.5f * (val1-val2) / (2.0f*maxval2 - val1 - val2);
            d_Orient[bx + maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        } else {
            d_Orient[bx + maxPts] = i2;
        }
    }
}