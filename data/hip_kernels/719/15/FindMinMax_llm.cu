#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];

    const int tx = threadIdx.x;
    const int x = blockIdx.x * blockDim.x + tx;
    const int y = blockIdx.y * blockDim.y;
    int p = y * pitch + x;

    float min_val = FLT_MAX;
    float max_val = -FLT_MAX;

    // Load data into registers to reduce global memory access
    for (int ty = 0; ty < 16 && (y + ty) < height; ty++) {
        if (x < width) {
            float val = d_Data[p];
            if (val < min_val) min_val = val;
            if (val > max_val) max_val = val;
        }
        p += pitch;
    }

    minvals[tx] = min_val;
    maxvals[tx] = max_val;
    __syncthreads();

    // Reduce within block
    for (int d = blockDim.x / 2; d > 0; d >>= 1) {
        if (tx < d) {
            if (minvals[tx + d] < minvals[tx]) minvals[tx] = minvals[tx + d];
            if (maxvals[tx + d] > maxvals[tx]) maxvals[tx] = maxvals[tx + d];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}