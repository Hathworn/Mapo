#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Calculate global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Calculate linear index
    int p = y * width + x;
    
    // Check bounds and perform computation
    if (x < width && y < height)
    {
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
    // No need for __syncthreads() in this context
}