#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;

    // Calculate stride for better access patterns
    const int stride = 16;

    for (int y = 0; y < h / 32; y++) {
        for (int x = tx; x < w; x += stride) {  // Use tx as a stride
            unsigned int val = minmax[y * w + x];

            if (val) {
                for (int k = 0; k < 32; k++) {
                    if ((val & 0x1) && (numPts < maxPts)) {
                        ptrs[stride * numPts + tx] = (y * 32 + k) * w + x;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}