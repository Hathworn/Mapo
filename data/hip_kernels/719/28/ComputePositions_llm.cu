#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputePositions(
    float *g_Data1, float *g_Data2, float *g_Data3, 
    int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
    int i = blockIdx.x * POSBLK_SIZE + threadIdx.x;
    if (i >= numPts) return;
    
    int p = d_Ptrs[i];
    float val[7];
    val[0] = g_Data2[p];
    val[1] = g_Data2[p-1];
    val[2] = g_Data2[p+1];

    // Precompute reused expressions to reduce operations
    float dx2 = 0.5f * (val[2] - val[1]);
    float dxx2 = 2.0f * val[0] - val[1] - val[2];
    
    val[3] = g_Data2[p-w];
    val[4] = g_Data2[p+w];
    
    float dy2 = 0.5f * (val[4] - val[3]);
    float dyy2 = 2.0f * val[0] - val[3] - val[4];
    
    val[5] = g_Data3[p];
    val[6] = g_Data1[p];
    
    float ds2 = 0.5f * (val[6] - val[5]);
    float dss2 = 2.0f * val[0] - val[5] - val[6];
    
    float dxy = 0.25f * (g_Data2[p+w+1] + g_Data2[p-w-1] - g_Data2[p-w+1] - g_Data2[p+w-1]);
    float dxs = 0.25f * (g_Data3[p+1] + g_Data1[p-1] - g_Data1[p+1] - g_Data3[p-1]);
    float dys = 0.25f * (g_Data3[p+w] + g_Data1[p-w] - g_Data3[p-w] - g_Data1[p+w]);

    // Reuse partial results to reduce computation
    float idxx = dyy2 * dss2 - dys * dys;
    float idxy = dys * dxs - dxy * dss2;
    float idxs = dxy * dys - dyy2 * dxs;
    float idyy = dxx2 * dss2 - dxs * dxs;
    float idys = dxy * dxs - dxx2 * dys;
    float idss = dxx2 * dyy2 - dxy * dxy;
    
    float det = idxx * dxx2 + idxy * dxy + idxs * dxs;
    float idet = 1.0f / det;
    
    // Use precomputed expressions
    float pdx = idet * (idxx * dx2 + idxy * dy2 + idxs * ds2);
    float pdy = idet * (idxy * dx2 + idyy * dy2 + idys * ds2);
    float pds = idet * (idxs * dx2 + idys * dy2 + idss * ds2);

    // Optimize condition check
    if (fabsf(pdx) > 0.5f || fabsf(pdy) > 0.5f || fabsf(pds) > 0.5f) {
        pdx = __fdividef(dx2, dxx2);
        pdy = __fdividef(dy2, dyy2);
        pds = __fdividef(ds2, dss2);
    }
    
    float dval = 0.5f * (dx2 * pdx + dy2 * pdy + ds2 * pds);
    d_Sift[i] = (p % w) + pdx;
    d_Sift[i + maxPts] = (p / w) + pdy;
    d_Sift[i + 2 * maxPts] = d_ConstantA[0] * exp2f(pds * d_ConstantB[0]);
    d_Sift[i + 3 * maxPts] = val[0] + dval;

    // Use precomputed trace and determinant
    float tra = dxx2 + dyy2;
    det = dxx2 * dyy2 - dxy * dxy;
    d_Sift[i + 4 * maxPts] = __fdividef(tra * tra, det);
}