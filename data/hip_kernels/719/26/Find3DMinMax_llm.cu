#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int height)
{
    // Improved cache by reducing redundant memory usage
    __shared__ float data1[(MINMAX_SIZE + 2) * 3];
    __shared__ float data2[(MINMAX_SIZE + 2) * 3];
    __shared__ float data3[(MINMAX_SIZE + 2) * 3];
    __shared__ float ymin[(MINMAX_SIZE + 2) * 3];
    __shared__ float ymax[(MINMAX_SIZE + 2) * 3];

    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_SIZE;
    const int xEndClamped = min(xStart + MINMAX_SIZE - 1, width - 1);
    const int xReadPos = xStart + tx - WARP_SIZE;
    const int xWritePos = xStart + tx;
    int memWid = MINMAX_SIZE + 2;

    int memPos0 = (tx - WARP_SIZE + 1);
    int memPos1 = (tx - WARP_SIZE + 1);
    int yq = 0;
    unsigned int output = 0;
    
    for (int y = 0; y < 34; y++) {
        output >>= 1;
        int memPos = yq * memWid + (tx - WARP_SIZE + 1);
        int yp = 32 * blockIdx.y + y - 1;
        yp = max(yp, 0);
        yp = min(yp, height - 1);
        int readStart = yp * width;

        // Reducing branch divergence and improving memory access
        if (tx >= (WARP_SIZE - 1) && xReadPos >= 0 && xReadPos < width) {
            data1[memPos] = d_Data1[readStart + xReadPos];
            data2[memPos] = d_Data2[readStart + xReadPos];
            data3[memPos] = d_Data3[readStart + xReadPos];
        } else {
            data1[memPos] = 0;
            data2[memPos] = 0;
            data3[memPos] = 0;
        }
        __syncthreads();

        int memPos2 = yq * memWid + tx;
        if (y > 1 && tx < memWid) {
            // Utilize shared memory efficiently
            float min1 = fminf(fminf(data1[memPos0], data1[memPos1]), data1[memPos2]);
            float min2 = fminf(fminf(data2[memPos0], data2[memPos1]), data2[memPos2]);
            float min3 = fminf(fminf(data3[memPos0], data3[memPos1]), data3[memPos2]);
            float max1 = fmaxf(fmaxf(data1[memPos0], data1[memPos1]), data1[memPos2]);
            float max2 = fmaxf(fmaxf(data2[memPos0], data2[memPos1]), data2[memPos2]);
            float max3 = fmaxf(fmaxf(data3[memPos0], data3[memPos1]), data3[memPos2]);

            ymin[tx] = fminf(fminf(min1, min2), min3);
            ymax[tx] = fmaxf(fmaxf(max1, max2), max3);
        }
        __syncthreads();

        if (y > 1 && tx < MINMAX_SIZE && xWritePos <= xEndClamped) {
            // Streamlined conditional checks
            float minv = fminf(ymin[tx], fminf(ymin[tx + 2], data2[memPos2 + 1]));
            minv = fminf(minv, d_ConstantA[1]);
            float maxv = fmaxf(ymax[tx], fmaxf(ymax[tx + 2], data2[memPos2 + 1]));
            maxv = fmaxf(maxv, d_ConstantA[0]);
            
            if (data2[memPos1 + 1] < minv || data2[memPos1 + 1] > maxv)
                output |= 0x80000000;
        }
        __syncthreads();

        memPos0 = memPos1;
        memPos1 = memPos2;
        yq = (yq < 2 ? yq + 1 : 0);
    }
    
    if (tx < MINMAX_SIZE && xWritePos < width) {
        int writeStart = blockIdx.y * width + xWritePos;
        d_Result[writeStart] = output;
    }
}