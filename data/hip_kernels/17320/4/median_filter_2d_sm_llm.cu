#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int clamp(int value, int bound)
{
    if (value < 0) return 1;
    if (value < bound) return value;
    return bound - 1;
}

__device__ int index(int x, int y, int width)
{
    return (y * width) + x;
}

__device__ const int FILTER_SIZE = 9;
__device__ const int FILTER_HALFSIZE = FILTER_SIZE >> 1;

__device__ void sort_bubble(float *x, int n_size)
{
    for (int i = 0; i < n_size - 1; i++)
    {
        for (int j = 0; j < n_size - i - 1; j++)
        {
            if (x[j] > x[j+1])
            {
                float temp = x[j];
                x[j] = x[j+1];
                x[j+1] = temp;
            }
        }
    }
}

__global__ void median_filter_2d_sm(unsigned char* input, unsigned char* output, int width, int height)
{
    __shared__ int sharedPixels[BLOCKDIM + FILTER_SIZE][BLOCKDIM + FILTER_SIZE];

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    int sharedIdxX = threadIdx.x + FILTER_HALFSIZE;
    int sharedIdxY = threadIdx.y + FILTER_HALFSIZE;
    int globalIdx = index(clamp(x, width), clamp(y, height), width);
    
    // Load pixels into shared memory
    sharedPixels[sharedIdxX][sharedIdxY] = input[globalIdx];
    
    // Ensure threads have all loaded their data before proceeding
    __syncthreads();

    // Compute the median filter only if within image bounds
    if ((x < width) && (y < height))
    {
        const int color_tid = y * width + x;
        float windowMedian[FILTER_SIZE * FILTER_SIZE];
        int windowElements = 0;

        #pragma unroll
        for (int x_iter = -FILTER_HALFSIZE; x_iter <= FILTER_HALFSIZE; x_iter++)
        {
            #pragma unroll
            for (int y_iter = -FILTER_HALFSIZE; y_iter <= FILTER_HALFSIZE; y_iter++)
            {
                if (0 <= x + x_iter && x + x_iter < width && 0 <= y + y_iter && y + y_iter < height)
                {
                    windowMedian[windowElements++] = sharedPixels[sharedIdxX + x_iter][sharedIdxY + y_iter];
                }
            }
        }
        
        sort_bubble(windowMedian, windowElements);
        output[color_tid] = windowMedian[windowElements / 2];
    }
}