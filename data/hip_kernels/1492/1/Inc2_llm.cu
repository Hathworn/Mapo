#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc2(float *Ad, float *Bd) {
    // Use shared memory for improved memory access times
    __shared__ float sharedA[LEN]; 

    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if(tx < 1024) {
        // Load data into shared memory
        sharedA[threadIdx.x] = Ad[tx];

        for(int i = 0; i < ITER; i++) {
            sharedA[threadIdx.x] += 1.0f;

            // Use a non-divergent loop for copying
            #pragma unroll // Unroll the loop for better performance
            for(int j = 0; j < 256; j++) {
                Bd[tx] = sharedA[threadIdx.x];
            }
        }

        // Write back to global memory
        Ad[tx] = sharedA[threadIdx.x];
    }
}