#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc1(float *Ad, float *Bd) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Avoid out of bounds access and unnecessary checks
    if (tx >= LEN) return;
    
    // Unroll loop to increase parallelism and efficiency
    float aValue = Ad[tx];
    for (int i = 0; i < ITER; ++i) {
        aValue += 1.0f;
    }
    Ad[tx] = aValue;
    
    // Reduce loop frequency by storing value to Bd once after computation
    Bd[tx] = aValue;
}