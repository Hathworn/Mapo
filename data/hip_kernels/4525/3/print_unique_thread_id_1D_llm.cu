#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_unique_thread_id_1D()
{
    // Calculate unique thread ID using 1D grid and block indices
    int tid = blockIdx.x * blockDim.x + threadIdx.x; 

    // Use a more efficient conditional check to print only if this thread is active
    if (tid < gridDim.x * blockDim.x) {
        printf("Thread{%d,%d,%d}, Block{%d,%d,%d}, tid{%d}\n", 
               threadIdx.x, threadIdx.y, threadIdx.z, 
               blockIdx.x, blockIdx.y, blockIdx.z, 
               tid);
    }
}