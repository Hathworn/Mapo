#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_cuda()
{
    // Use thread and block indices for potential expanded functionality 
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Print from a single, identifiable thread to reduce output clutter
    if (idx == 0) {
        printf("Hello Cuda\n");
    }
}