#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_thread_variables()
{
    // Use a local buffer to reduce printf frequency and minimize potential bottleneck
    char buffer[256];
    snprintf(buffer, sizeof(buffer), "Thread{%d,%d,%d}, Block{%d,%d,%d}, BlockDim{%d,%d,%d}, GridDim{%d,%d,%d}\n",
             threadIdx.x, threadIdx.y, threadIdx.z,
             blockIdx.x, blockIdx.y, blockIdx.z,
             blockDim.x, blockDim.y, blockDim.z,
             gridDim.x, gridDim.y, gridDim.z);

    // Output the formatted string
    printf("%s", buffer);
}