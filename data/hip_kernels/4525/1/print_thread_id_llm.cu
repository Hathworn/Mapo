#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_thread_id()
{
    // Use variables to store the calculated thread ID for better clarity and future optimizations.
    int tid_x = threadIdx.x + blockDim.x * blockIdx.x;
    int tid_y = threadIdx.y + blockDim.y * blockIdx.y;
    int tid_z = threadIdx.z + blockDim.z * blockIdx.z;

    // Print the thread ID in a concise manner.
    printf("Hello Cuda tid[%d, %d, %d]\n", tid_x, tid_y, tid_z);
}