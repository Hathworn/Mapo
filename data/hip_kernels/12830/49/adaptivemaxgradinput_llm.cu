#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w)
{
    // Compute offsets for parallel execution based on thread and block ID
    int o = blockIdx.x;
    int i = o;
    gradOutput = gradOutput + o * output_w * output_h;
    gradInput = gradInput + i * input_w * input_h;
    indices_x = indices_x + o * output_w * output_h;
    indices_y = indices_y + o * output_w * output_h;

    int xx_start = threadIdx.x + blockIdx.y * blockDim.x;
    int xx_step = blockDim.x * gridDim.y;
    int yy_start = threadIdx.y;
    int yy_step = blockDim.y;

    // Compute gradInput using efficient thread indexing
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        int y_start = (int)floor(float(yy) / output_h * input_h);

        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            int x_start = (int)floor(float(xx) / output_w * input_w);

            float *ptr_gradInput = gradInput + y_start * input_w + x_start;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float *ptr_ind_x = indices_x + yy * output_w + xx;
            float *ptr_ind_y = indices_y + yy * output_w + xx;
            float z = *ptr_gradOutput;

            int argmax_x = (*ptr_ind_x) - 1;
            int argmax_y = (*ptr_ind_y) - 1;

            ptr_gradInput[argmax_x + argmax_y * input_w] += z;
        }
    }
}