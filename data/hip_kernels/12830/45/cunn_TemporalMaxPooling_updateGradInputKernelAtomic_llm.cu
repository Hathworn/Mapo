#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Pre-calculate commonly used values outside of loop
    int global_idx = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    if (global_idx < output_w) {
        float *gradInput_data = gradInput + blockIdx.x * input_w * input_n + global_idx * input_n * dW;
        float *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + global_idx * input_n;
        float *indices_data = indices + blockIdx.x * output_w * input_n + global_idx * input_n;

        // Unrolling loop for better performance
        for (int feat = 0; feat < input_n; feat += 4) {
            atomicAdd(&gradInput_data[(int)indices_data[feat] * input_n + feat], gradOutput_data[feat]);
            if (feat + 1 < input_n) atomicAdd(&gradInput_data[(int)indices_data[feat + 1] * input_n + feat + 1], gradOutput_data[feat + 1]);
            if (feat + 2 < input_n) atomicAdd(&gradInput_data[(int)indices_data[feat + 2] * input_n + feat + 2], gradOutput_data[feat + 2]);
            if (feat + 3 < input_n) atomicAdd(&gradInput_data[(int)indices_data[feat + 3] * input_n + feat + 3], gradOutput_data[feat + 3]);
        }
    }
}