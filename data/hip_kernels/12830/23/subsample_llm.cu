#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;

    int yy_start = blockIdx.y * blockDim.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Offset the input and output pointers
    output += o * output_w * output_h;
    input += o * input_w * input_h;

    // Load the weight and bias for the current output/input plane
    float the_weight = weight[k];
    float the_bias = bias[k];

    // Loop through output pixels
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            // Compute sum of the input image section
            float sum = 0.0f;
            int input_idx = (yy * dH) * input_w + xx * dW;
            for (int ky = 0; ky < kH; ++ky) {
                for (int kx = 0; kx < kW; ++kx) {
                    sum += input[input_idx + kx];
                }
                input_idx += input_w; // Move to the next row in the input
            }
            // Compute and store the output value
            output[yy * output_w + xx] = the_weight * sum + the_bias;
        }
    }
}