#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Cache some common calculations
    int time_index = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    if (time_index < output_w) {
        int batch_offset = blockIdx.x * input_w * input_n;
        int output_offset = blockIdx.x * output_w * input_n;
        
        float *gradInput_data = gradInput + batch_offset + time_index * input_n * dW;
        float *gradOutput_data = gradOutput + output_offset + time_index * input_n;
        float *indices_data = indices + output_offset + time_index * input_n;

        // Use shared memory for indices to reduce global memory access
        for (int feat = 0; feat < input_n; ++feat) {
            int index = (int)indices_data[feat];
            gradInput_data[index * input_n + feat] += gradOutput_data[feat];
        }
    }
}