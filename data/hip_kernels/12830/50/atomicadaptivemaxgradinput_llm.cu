#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void atomicadaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w) {
    // Compute global thread indices
    int o = blockIdx.x;
    int xx = blockIdx.y * blockDim.x + threadIdx.x;
    int yy = blockIdx.z * blockDim.y + threadIdx.y;

    if (xx < output_w && yy < output_h) {
        // Select input/output plane directly
        gradOutput += o * output_w * output_h;
        gradInput += o * input_w * input_h;
        indices_x += o * output_w * output_h;
        indices_y += o * output_w * output_h;

        int y_start = static_cast<int>(floorf(float(yy) / output_h * input_h));
        int x_start = static_cast<int>(floorf(float(xx) / output_w * input_w));

        float *ptr_gradInput = gradInput + y_start * input_w + x_start;
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float *ptr_ind_x = indices_x + yy * output_w + xx;
        float *ptr_ind_y = indices_y + yy * output_w + xx;
        float z = *ptr_gradOutput;

        int argmax_x = static_cast<int>(*ptr_ind_x) - 1;
        int argmax_y = static_cast<int>(*ptr_ind_y) - 1;

        // Atomic addition
        atomicAdd(&(ptr_gradInput[argmax_x + argmax_y * input_w]), z);
    }
}