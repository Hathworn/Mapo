#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Calculate offsets
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    // Calculate start position and step size for each thread
    int xx_start = threadIdx.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int xx_step = blockDim.x;
    int yy_step = blockDim.y * gridDim.y;

    // Pointer arithmetic for plane selection
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;

    // Get the weight for the current index
    float the_weight = weight[k];

    // Calculate gradInput using atomic operations
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = (*ptr_gradOutput) * the_weight;
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    atomicAdd(&(ptr_gradInput[kx]), z); // Atomic addition to avoid race condition
                }
                ptr_gradInput += input_w; // Move to the next row in the input
            }
        }
    }
}