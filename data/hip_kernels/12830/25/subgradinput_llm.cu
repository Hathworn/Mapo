#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradinput(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Determine offsets and indices based on thread and block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Offset pointers to the start of the correct output and input planes
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;

    // Load the weight once per block
    float the_weight = weight[k];

    // Main loop optimized for memory coalescing and reduced global memory access
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput * the_weight;
            // Propagate the gradient to the input
            for (int ky = 0; ky < kH; ++ky) {
                for (int kx = 0; kx < kW; ++kx) {
                    ptr_gradInput[kx] += z;
                }
                ptr_gradInput += input_w;
            }
        }
    }
}