#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bf_2flags(int *Na, int *src, int *F1, int *F2, int *exists, int *Sa, int *Ea, int threadsPerBlock)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim.x for dimension

    if (exists[id] == 1)
    {
        Na[id] = 65000; // MAX INT Value
        F1[id] = 0;
        F2[id] = 0;

        if (id == *src)
        { // Starting node conditions
            Na[id] = 0;
            F1[id] = 1;
        }

        // Use a shared memory flag for synchronization
        // __shared__ int localF1[blockDim.x];
        // __shared__ int localF2[blockDim.x];

        for (int i = 0; i < 103689; ++i)
        {
            if (F1[Sa[id]] == 1)
            {
                // Replace atomic operation with direct update (not thread-safe)
                if (Na[Ea[id]] > Na[Sa[id]] + 1)
                {
                    Na[Ea[id]] = Na[Sa[id]] + 1;
                    F2[Ea[id]] = 1;
                }
            }

            // Swap and reset flags using temporary variable
            int temp = F1[id];
            F1[id] = F2[id];
            F2[id] = temp;
        }
    }
}