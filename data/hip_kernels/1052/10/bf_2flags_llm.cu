#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bf_2flags(int *Na, int *src, int *F1, int *F2, int *exists, int *Sa, int *Ea, int threadsPerBlock )
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim.x for threadsPerBlock

    if (exists[id] == 1)
    {
        Na[id] = 65000; // MAX INT Value
        F1[id] = 0;
        F2[id] = 0;

        if (id == *src)
        {   // Starting node conditions
            Na[id] = 0;
            F1[id] = 1;
        }

        // Loop optimizations
        __shared__ int sharedF1[1024]; // Use shared memory for F1
        __shared__ int sharedF2[1024]; // Use shared memory for F2

        sharedF1[threadIdx.x] = F1[id];
        sharedF2[threadIdx.x] = F2[id];

        __syncthreads(); // Synchronize threads

        for (int i = 0; i < 103689; ++i)
        {
            if (sharedF1[threadIdx.x] == 1)
            {
                if (Na[Ea[id]] > Na[Sa[id]] + 1)
                {
                    // Relax
                    Na[Ea[id]] = Na[Sa[id]] + 1;
                    sharedF2[threadIdx.x] = 1;
                }
            }

            // Swap and update flags with synchronization
            __syncthreads();
            F1[id] = sharedF2[threadIdx.x];
            sharedF1[threadIdx.x] = sharedF2[threadIdx.x];
            sharedF2[threadIdx.x] = 0;
            __syncthreads();
        }
    }
}