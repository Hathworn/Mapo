#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rotateArray(int *c, int numThreads)
{
    // Compute the next index for each thread using blockIdx and blockDim for more flexibility
    int nextIndex = (blockIdx.x * blockDim.x + threadIdx.x + 1) % numThreads;

    // Load value from next index
    int val = c[nextIndex];

    // Ensure that all threads have performed their read before writing
    __syncthreads();

    // Write value to current index
    c[blockIdx.x * blockDim.x + threadIdx.x] = val;
}