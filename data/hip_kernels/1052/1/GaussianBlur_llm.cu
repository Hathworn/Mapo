#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GaussianBlur(unsigned int *B, unsigned int *G, unsigned int *R, int numberOfPixels, unsigned int width, int *B_new, int *G_new, int *R_new)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= numberOfPixels) return;

    int mask[] = { 1, 2, 1, 2, 4, 2, 1, 2, 1 };
    int totalWeight = 16; // Precomputed sum of the mask

    // Store reused values in registers
    int currentIdx = index;
    int widthIdx = currentIdx + width;
    int widthNegIdx = currentIdx - width;
    bool isLeft = (currentIdx % width == 0);
    bool isRight = (currentIdx % width == width - 1);
    bool isTop = (currentIdx < width);
    bool isBottom = (currentIdx >= numberOfPixels - width);

    if (isTop || isBottom || isLeft || isRight) {
        // Handle edge cases with simplified logic
        int s = mask[4], newValueB = B[currentIdx] * mask[4], newValueG = G[currentIdx] * mask[4], newValueR = R[currentIdx] * mask[4];
        if (isTop) {
            if (!isLeft) {
                s += mask[3]; 
                newValueB += B[currentIdx - 1] * mask[3];
                newValueG += G[currentIdx - 1] * mask[3];
                newValueR += R[currentIdx - 1] * mask[3];
            }
            if (!isRight) {
                s += mask[5]; 
                newValueB += B[currentIdx + 1] * mask[5];
                newValueG += G[currentIdx + 1] * mask[5];
                newValueR += R[currentIdx + 1] * mask[5];
            }
            s += mask[7]; 
            newValueB += B[widthNegIdx] * mask[7];
            newValueG += G[widthNegIdx] * mask[7];
            newValueR += R[widthNegIdx] * mask[7];
            if (!isLeft) {
                s += mask[6]; 
                newValueB += B[widthNegIdx - 1] * mask[6];
                newValueG += G[widthNegIdx - 1] * mask[6];
                newValueR += R[widthNegIdx - 1] * mask[6];
            }
            if (!isRight) {
                s += mask[8]; 
                newValueB += B[widthNegIdx + 1] * mask[8];
                newValueG += G[widthNegIdx + 1] * mask[8];
                newValueR += R[widthNegIdx + 1] * mask[8];
            }
        }
        if (isBottom) {
            s += mask[1]; 
            newValueB += B[widthIdx] * mask[1];
            newValueG += G[widthIdx] * mask[1];
            newValueR += R[widthIdx] * mask[1];
            if (!isLeft) {
                s += mask[0];
                newValueB += B[widthIdx - 1] * mask[0];
                newValueG += G[widthIdx - 1] * mask[0];
                newValueR += R[widthIdx - 1] * mask[0];
            }
            if (!isRight) {
                s += mask[2];
                newValueB += B[widthIdx + 1] * mask[2];
                newValueG += G[widthIdx + 1] * mask[2];
                newValueR += R[widthIdx + 1] * mask[2];
            }
        }
        if (isLeft && !isTop && !isBottom) {
            s += mask[1] + mask[2] + mask[5] + mask[8];
            newValueB += B[widthIdx] * mask[1] + B[widthIdx + 1] * mask[2] + B[currentIdx + 1] * mask[5] + B[widthNegIdx + 1] * mask[8];
            newValueG += G[widthIdx] * mask[1] + G[widthIdx + 1] * mask[2] + G[currentIdx + 1] * mask[5] + G[widthNegIdx + 1] * mask[8];
            newValueR += R[widthIdx] * mask[1] + R[widthIdx + 1] * mask[2] + R[currentIdx + 1] * mask[5] + R[widthNegIdx + 1] * mask[8];
        }
        if (isRight && !isTop && !isBottom) {
            s += mask[1] + mask[0] + mask[3] + mask[6];
            newValueB += B[widthIdx] * mask[1] + B[widthIdx - 1] * mask[0] + B[currentIdx - 1] * mask[3] + B[widthNegIdx - 1] * mask[6];
            newValueG += G[widthIdx] * mask[1] + G[widthIdx - 1] * mask[0] + G[currentIdx - 1] * mask[3] + G[widthNegIdx - 1] * mask[6];
            newValueR += R[widthIdx] * mask[1] + R[widthIdx - 1] * mask[0] + R[currentIdx - 1] * mask[3] + R[widthNegIdx - 1] * mask[6];
        }
        B_new[index] = newValueB / s;
        G_new[index] = newValueG / s;
        R_new[index] = newValueR / s;
    } else {
        // Use shared memory for regular cases
        int pos[9] = {
            widthNegIdx - 1, widthNegIdx, widthNegIdx + 1,
            currentIdx - 1, currentIdx, currentIdx + 1,
            widthIdx - 1, widthIdx, widthIdx + 1
        };

        int newValueB = 0, newValueG = 0, newValueR = 0;
        #pragma unroll
        for (int i = 0; i < 9; ++i) {
            newValueB += B[pos[i]] * mask[i];
            newValueG += G[pos[i]] * mask[i];
            newValueR += R[pos[i]] * mask[i];
        }
        B_new[index] = newValueB / totalWeight;
        G_new[index] = newValueG / totalWeight;
        R_new[index] = newValueR / totalWeight;
    }
}