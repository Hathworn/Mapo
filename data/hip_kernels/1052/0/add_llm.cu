#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*

Jaitirth Jacob - 13CO125      Vidit Bhargava - 13CO151

*/

#define ITERATIONS 4	//Repeat the experiment for greater accuracy

#define N 1000000		//Array Size
#define min_threads 16
#define max_threads 1024

__global__ void add(int *a, int *b, int *c, int tpb)
{
    // Calculate thread index only if within bounds for safety
    int i = blockIdx.x * tpb + threadIdx.x;
    if (i < N) {
        c[i] = a[i] + b[i]; // Perform addition
    }
}