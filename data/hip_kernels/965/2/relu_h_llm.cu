#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_h(float *X, float *Y, int size_in) {
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    if (t < size_in) {
        // Use max function for conditional assignment
        Y[t] = max(0.0f, X[t]);
    }
}