#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel using shared memory
__global__ void reduce_to_first_index_h(float *X, int height, int width) {
    extern __shared__ float shared_data[];
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load data into shared memory
    if (t < width) {
        shared_data[threadIdx.x] = 0;
        for (int i = 0; i < height; i++) {
            shared_data[threadIdx.x] += X[i * width + t];
        }
        
        // Synchronize threads before writing back
        __syncthreads();

        // Write back the result to global memory
        if (threadIdx.x == 0) {
            for (int j = 1; j < blockDim.x; j++) {
                shared_data[0] += shared_data[j];
            }
            X[t] = shared_data[0];
        }
    }
}