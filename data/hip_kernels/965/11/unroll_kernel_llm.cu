#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unroll_kernel(int h_in, int w_in, int k, float *x, float *x_unroll) {
    int t = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim.x for flexibility
    int w_out = w_in - k + 1;                      // Output image size
    int w_unroll = w_out * w_out;                  // Unroll limit

    if (t < w_unroll) {
        int h_out_ = t / w_out;                    // Output height
        int w_out_ = t % w_out;                    // Output width
        int w_unroll_ = h_out_ * w_out + w_out_;   // The index of output pixel in image
        int h_unroll;

        #pragma unroll                            // Unroll loop for efficiency
        for (int p = 0; p < k; p++) {
            #pragma unroll                        // Unroll loop for efficiency
            for (int q = 0; q < k; q++) {
                h_unroll = p * k + q;
                if ((h_out_ + p) < h_in && (w_out_ + q) < w_in) {
                    x_unroll[h_unroll * w_unroll + w_unroll_] =
                    x[(h_out_ + p) * w_in + w_out_ + q];
                }
            }
        }
    }
}