#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void grad_descent(float *odata, const float *idata, int size) {
    // Optimize using dynamic shared memory
    extern __shared__ float shared_data[];
    
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (t < size) {
        // Load data into shared memory
        shared_data[threadIdx.x] = idata[t];
        __syncthreads();

        // Perform calculation using shared memory
        odata[t] -= LEARNIG_RATE * shared_data[threadIdx.x];
    }
}