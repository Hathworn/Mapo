#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_relu_h(float *X, float *Y, int size_in) {
    int t = blockIdx.x * blockDim.x + threadIdx.x;

    // Simplifying the conditional operation using a ternary operator
    if (t < size_in) {
        X[t] = (X[t] >= 0) ? Y[t] : 0.0;
    }
}