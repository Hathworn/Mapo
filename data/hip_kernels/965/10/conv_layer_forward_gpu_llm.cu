#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv_layer_forward_gpu(float *x, float *w, float *y, int h_in, int w_in, int w_out, int k, int m) {
    int n = blockIdx.x;   // Batch index
    int m_ = blockIdx.y;  // Channel index
    int h = threadIdx.y;  // Pixel (h, w_)
    int w_ = threadIdx.x; // Pixel (h, w_)
    float ans = 0;        // Return value
    int offset = n * (h_in * w_in);

    // Load w into shared memory to speed up the data access
    __shared__ float cached_w[CONV_KERNEL_SIZE][CONV_KERNEL_SIZE];
    if (h < k && w_ < k) {
        cached_w[h][w_] = w[m_ * (k * k) + h * k + w_];
    }
    __syncthreads();

    // Loop over k by k kernel
    if (h < w_out && w_ < w_out) {
        #pragma unroll   // Unroll the inner loop for better performance
        for (int p = 0; p < k; p++) {
            #pragma unroll   // Unroll the inner loop for better performance
            for (int q = 0; q < k; q++) {
                ans += x[offset + (h + p) * w_in + (w_ + q)] * cached_w[p][q];
            }
        }
        // Write out the return value
        y[n * (m * w_out * w_out) + m_ * (w_out * w_out) + h * w_out + w_] = ans;
    }
}