#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronized blocks - optimized
__global__ void copy_0(int nx, int ny, int nz, Real *in, Real *out) {
    int tj = threadIdx.x + blockIdx.x * blockDim.x; // Simplify computation by including blockIdx.x
    int index_in = blockIdx.y * nx * ny / 4 + tj; // Single index for input
    int index_out = (blockIdx.y + nz / 2) * nx * ny + tj + ny / 2 * nx; // Single index for output

    // Perform memory access, assuming boundaries are correctly managed
    if (tj < nx * ny / 4) {
        out[index_out] = in[index_in];
    }
}