#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
__global__ void copy_0(int nx, int ny, int nz, Real *in, Real *out) {

int tj = threadIdx.x;
int td = blockDim.x;

int jj =  (blockIdx.y*nx*ny/4 + blockIdx.x*nx/2);
int jj1 =  ((blockIdx.y+nz/2)*nx*ny + (blockIdx.x+ny/2)*nx);


out[jj1+tj+td] = in[jj+tj];

}