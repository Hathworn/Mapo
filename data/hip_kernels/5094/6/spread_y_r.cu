#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
__global__ void spread_y_r(Real* src, Real* dst)
{
unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
unsigned int tid1 = (blockIdx.y * gridDim.x * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

Real res =  src[tid];
dst[tid1 + blockDim.x*gridDim.x] = res;
#ifdef DOUBLE
dst[tid1] = 0.;
#else
dst[tid1] = 0.f;
#endif
}