#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// optimize by using shared memory where applicable
__global__ void spread_y_r(Real* __restrict__ src, Real* __restrict__ dst)
{
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;
    unsigned int tid1 = tid + blockDim.x * gridDim.x * blockIdx.y;

    Real res = src[tid];
    dst[tid1 + blockDim.x * gridDim.x] = res;
#ifdef DOUBLE
    dst[tid1] = 0.;
#else
    dst[tid1] = 0.f;
#endif
}