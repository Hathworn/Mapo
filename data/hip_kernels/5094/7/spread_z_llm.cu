#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM 8

// Optimized kernel to spread values
__global__ void spread_z(Real* src, Real* dst)
{
    // Calculate global thread ID
    unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    // Directly access and modify the source array
    Real res = src[tid];
    src[tid + (gridDim.y * gridDim.x) * blockDim.x] = res; // Efficient indexing
#ifdef DOUBLE
    src[tid] = 0.0; // Set value
#else
    src[tid] = 0.f; // Set value
#endif
}