#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
__global__ void spread_z(Real* src, Real* dst)
{
unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
//unsigned int tid1 = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

Real res =  src[tid];
src[tid + (gridDim.y * gridDim.x) * blockDim.x] = res;
#ifdef DOUBLE
src[tid] = 0.0;
#else
src[tid] = 0.f;
#endif
}