#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
__global__ void zero(int nx, int ny, int nz, Real *z) {
    int tj = threadIdx.x;
    int td = blockDim.x;
    int blockData = (nx * ny * nz) / (gridDim.x * gridDim.y);
    int jj = ((blockIdx.y) * gridDim.x + (blockIdx.x)) * blockData;

    // Calculate start index for each thread
    int start_idx = jj + tj;
    // Unroll the loop for better performance and to reduce loop overhead
    #pragma unroll
    for (int k = 0; k < blockData / td; ++k) {
        z[start_idx + k * td] = 0.0;
    }
}