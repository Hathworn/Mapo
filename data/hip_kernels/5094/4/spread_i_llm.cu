#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// Optimize global kernel
__global__ void spread_i(Real* src, unsigned int spitch, Real* dst, unsigned int dpitch) 
{
    unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int tid = threadIdx.x;

    // Use shared memory for coalesced memory access
    __shared__ Real sharedData[TILE_DIM];

    if (tid < dpitch) {
        // Coalesce reads from src into shared memory
        sharedData[tid] = src[bid * dpitch + tid];
    }
    __syncthreads();

    if (tid < dpitch) {
        // Coalesce writes from shared memory to dst
        dst[bid * spitch + tid] = sharedData[tid];
    }
}