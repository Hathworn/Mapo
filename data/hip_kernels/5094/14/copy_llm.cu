#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
__global__ void copy(int nx, int ny, int nz, Real *in, Real *out) {
    int tj = threadIdx.x;

    int jj = (blockIdx.y * nx * ny / 4 + blockIdx.x * nx / 2);
    int jj1 = (blockIdx.y * nx * ny + blockIdx.x * nx);

    // Coalesce global memory access by using aligned indexing.
    if (tj < TILE_DIM) {
        out[jj + tj] = in[jj1 + tj];
    }
}