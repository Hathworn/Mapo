#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
__global__ void spread_y_i_r(Real* src, Real* dst)
{
    // Efficient tile-based indexing
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idy = blockIdx.y * gridDim.x * blockDim.x;
    
    unsigned int tid = idy + idx;
    unsigned int tid1 = idy * 2 + idx;

    // Load from contiguous memory location
    dst[tid] = src[tid1];
}