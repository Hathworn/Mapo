#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
__global__ void spread_y_i_r(Real* src, Real* dst)
{
unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
unsigned int tid1 = (blockIdx.y * gridDim.x * 2 + blockIdx.x) * blockDim.x + threadIdx.x;

Real res =  src[tid1];
dst[tid] = res;
}