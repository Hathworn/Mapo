#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
__global__ void spread(Real* src, unsigned int spitch, Real* dst, unsigned int dpitch)
{
unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
unsigned int tid = threadIdx.x;

Real res = (tid >= spitch) ? src[bid * spitch + tid-spitch] : 0.0;
if( tid < dpitch) {
dst[bid * dpitch + tid] = res;
}
}