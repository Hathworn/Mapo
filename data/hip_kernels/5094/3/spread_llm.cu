#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
__global__ void spread(Real* __restrict__ src, unsigned int spitch, Real* __restrict__ dst, unsigned int dpitch)
{
    unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int tid = threadIdx.x;

    // Load only if within bounds, using ternary conditional within array calculation for efficiency
    Real res = (tid < spitch) ? src[bid * spitch + tid] : 0.0;

    // Store only if within bounds
    if(tid < dpitch) {
        dst[bid * dpitch + tid] = res;
    }
}