#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define TILE_WIDTH 32

#define COMMENT "Centrist_GPU"
#define RGB_COMPONENT_COLOR 255

typedef struct {
    unsigned char red, green, blue;
} PPMPixel;

typedef struct {
    int x, y;
    PPMPixel *data;
} PPMImage;

__global__ void mod_CENTRIST(PPMPixel *image_out, PPMPixel *image_cp, int columns, int rows, int *hist, int hist_len) {
    int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

    __shared__ int hist_private[512];
    int hist_index = (threadIdx.y * TILE_WIDTH + threadIdx.x); //get index in shared histogram
    if(hist_index < hist_len) hist_private[hist_index] = 0;
    __syncthreads();

    if(col < columns && row < rows) {
        // Use shared memory efficiently for image processing
        __shared__ unsigned char image_cp_private[TILE_WIDTH + 2][TILE_WIDTH + 2];
        
        int img_index = row * columns + col; //get index in original image
        int grayscale = (image_cp[img_index].red*299 + image_cp[img_index].green*587 + image_cp[img_index].blue*114) / 1000; //avoid float point errors

        image_cp_private[threadIdx.y + 1][threadIdx.x + 1] = grayscale;
        if(threadIdx.x == 0 && col > 0) {
            int left_grayscale = (image_cp[img_index - 1].red*299 + image_cp[img_index - 1].green*587 + image_cp[img_index - 1].blue*114) / 1000;
            image_cp_private[threadIdx.y + 1][threadIdx.x] = left_grayscale;
        }
        if(threadIdx.x == TILE_WIDTH - 1 && col < columns - 1) {
            int right_grayscale = (image_cp[img_index + 1].red*299 + image_cp[img_index + 1].green*587 + image_cp[img_index + 1].blue*114) / 1000;
            image_cp_private[threadIdx.y + 1][threadIdx.x + 2] = right_grayscale;
        }
        if(threadIdx.y == 0 && row > 0) {
            int top_grayscale = (image_cp[img_index - columns].red*299 + image_cp[img_index - columns].green*587 + image_cp[img_index - columns].blue*114) / 1000;
            image_cp_private[threadIdx.y][threadIdx.x + 1] = top_grayscale;
        }
        if(threadIdx.y == TILE_WIDTH - 1 && row < rows - 1) {
            int bottom_grayscale = (image_cp[img_index + columns].red*299 + image_cp[img_index + columns].green*587 + image_cp[img_index + columns].blue*114) / 1000;
            image_cp_private[threadIdx.y + 2][threadIdx.x + 1] = bottom_grayscale;
        }
        __syncthreads();

        if(col < columns - 1 && row < rows - 1) { //ignore first/last row/column
            float mean = 0.0f;
            for(int r = 0; r < 3; ++r) {
                for(int c = 0; c < 3; ++c) {
                    mean += image_cp_private[threadIdx.y + r][threadIdx.x + c];
                }
            }
            mean /= 9.0f;
            int value = 0, k = 8;
            for(int r = 0; r < 3; ++r) {
                for(int c = 0; c < 3; ++c) {
                    if(image_cp_private[threadIdx.y + r][threadIdx.x + c] >= mean) {
                        value |= 1 << k;
                    }
                    k--;
                }
            }

            int img_out_ind = row * (columns - 1) + col; //get index in output original
            image_out[img_out_ind].red = image_out[img_out_ind].blue = image_out[img_out_ind].green = value;
            atomicAdd(&(hist_private[value]), 1);
        }
        __syncthreads();

        if(hist_index == 0) {
            for(int i = 0; i < hist_len; i++) {
                atomicAdd(&(hist[i]), hist_private[i]); //init shared histogram
            }
        }
    }
}