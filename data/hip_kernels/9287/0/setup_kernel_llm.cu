#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    // Calculate global ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize random state
    hiprand_init(seed, id, 0, &state[id]);
}