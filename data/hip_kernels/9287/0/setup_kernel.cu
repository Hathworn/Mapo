#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_kernel(  hiprandState * state, unsigned long seed )
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
hiprand_init( seed, id, 0, &state[ id ] );

return;
}