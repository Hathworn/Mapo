#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computePositionParallel(float *agentsX, float *agentsY, float *destX, float *destY, float *destR, int n, int *reached) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        // Early exit if no destination
        if (destX[i] == -1 || destY[i] == -1) {
            continue;
        }

        // Precompute differences and check zero length to avoid division by zero
        double diffX = destX[i] - agentsX[i];
        double diffY = destY[i] - agentsY[i];
        double length = sqrtf(diffX * diffX + diffY * diffY);

        // Avoid division by zero
        if (length > 0) {
            double invLength = 1.0 / length;
            agentsX[i] = (float)llrintf(agentsX[i] + diffX * invLength);
            agentsY[i] = (float)llrintf(agentsY[i] + diffY * invLength);
        }

        // Recompute differences and check if inside radius
        diffX = destX[i] - agentsX[i];
        diffY = destY[i] - agentsY[i];
        length = sqrtf(diffX * diffX + diffY * diffY);

        if (length < destR[i]) {
            reached[i] = 1;
        }
    }
}