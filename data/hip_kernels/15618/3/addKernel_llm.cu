#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if (i < N) // Check bounds to avoid accessing invalid memory
    {
        c[i] = a[i] + b[i];
    }
}