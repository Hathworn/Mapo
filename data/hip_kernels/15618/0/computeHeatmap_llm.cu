#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define WEIGHTSUM 273
#define BLOCK_SIZE 16

int * heatmap;
size_t heatmap_pitch;

int * scaled_heatmap;
size_t scaled_heatmap_pitch;

int * blurred_heatmap;
size_t blurred_heatmap_pitch;

float* d_desiredPositionX;
float* d_desiredPositionY;

__global__ void computeHeatmap(float* desiredAgentsX, float* desiredAgentsY, int n, int* heatmap, size_t heatmap_pitch, int* scaled_heatmap, size_t scaled_heatmap_pitch) {
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Thread row and column block
    int row = threadIdx.y;
    int col = threadIdx.x;

    // x, y coordinate
    int x = blockCol * blockDim.x + col;
    int y = blockRow * blockDim.y + row;

    // Fade heatmap
    int* heatPoint = (int*)((char*)heatmap + y * heatmap_pitch) + x;
    *heatPoint = static_cast<int>(roundf((*heatPoint) * 0.80f));

    // Pull desiredAgentX and Y array from global to shared memory
    extern __shared__ float desiredPosition[];
    int tId = row * blockDim.x + col;

    for (int i = tId; i < n; i += blockDim.x * blockDim.y) {
        desiredPosition[i] = desiredAgentsX[i];
        desiredPosition[i + n] = desiredAgentsY[i];
    }

    __syncthreads();

    // Count how many agents want to go to each location
    for (int i = 0; i < n; i++) {
        int desiredX = static_cast<int>(desiredPosition[i]);
        int desiredY = static_cast<int>(desiredPosition[i + n]);

        if (x == desiredX && y == desiredY) {
            // Intensify heat for better color results
            atomicMin(heatPoint, 255 - 40);
            atomicAdd(heatPoint, 40);
        }
    }
}