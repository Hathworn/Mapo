#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define WEIGHTSUM 273
#define BLOCK_SIZE 16

int * heatmap;
size_t heatmap_pitch;

int * scaled_heatmap;
size_t scaled_heatmap_pitch;

int * blurred_heatmap;
size_t blurred_heatmap_pitch;

float* d_desiredPositionX;
float* d_desiredPositionY;

__global__ void computeScaledHeatmap(int* heatmap, size_t heatmap_pitch, int* scaled_heatmap, size_t scaled_heatmap_pitch) {
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Thread row and column within block
    int row = threadIdx.y;
    int col = threadIdx.x;

    // x, y coordinate
    int x = blockCol * blockDim.x + col;
    int y = blockRow * blockDim.y + row;

    // Scale the data for visual representation
    int value = *((int*)((char*)heatmap + y * heatmap_pitch) + x);

    // Reduce pitch calculations and avoid repeated casting
    char* base_s_row = (char*)scaled_heatmap + y * CELLSIZE * scaled_heatmap_pitch;
    for (int r = 0; r < CELLSIZE; r++) {
        int* row = (int*)(base_s_row + r * scaled_heatmap_pitch);
        for (int c = 0; c < CELLSIZE; c++) {
            row[x * CELLSIZE + c] = value; // Duplicate the value
        }
    }
}