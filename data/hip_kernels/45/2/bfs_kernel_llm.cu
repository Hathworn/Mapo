#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifndef _KERNEL_H
#define _KERNEL_H
typedef struct Node {
int starting;
int no_of_edges;
}Node;

#endif

__global__ void bfs_kernel(Node* d_graph_nodes, int* d_edge_list, bool* d_graph_level, bool* d_graph_visited, int* d_cost, bool* loop, int no_of_nodes) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit if thread index is out of bounds
    if (tid >= no_of_nodes) return;

    // Check if the vertex is in the current level and needs to be visited
    if (d_graph_level[tid]) {
        d_graph_level[tid] = false;
        d_graph_visited[tid] = true;

        // Cache graph node data in local variables for memory access optimization
        int start = d_graph_nodes[tid].starting;
        int end = start + d_graph_nodes[tid].no_of_edges;

        for (int i = start; i < end; i++) {
            int id = d_edge_list[i];

            // Check if the node is already visited
            if (!d_graph_visited[id]) {
                // Update cost and set the vertex to be visited in the next level
                d_cost[id] = d_cost[tid] + 1;
                d_graph_level[id] = true;

                // Signal that another iteration is needed
                *loop = true;
            }
        }
    }
}