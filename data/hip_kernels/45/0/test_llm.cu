#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifndef _KERNEL_H
#define _KERNEL_H
typedef struct Node {
    int starting;
    int no_of_edges;
} Node;

#endif

__global__ void test(Node* d_graph_nodes, int no_of_nodes) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < no_of_nodes) {
        // Use shared memory to minimize global memory access
        extern __shared__ Node shared_nodes[];
        Node temp_node = d_graph_nodes[tid];
        temp_node.starting += 1;
        shared_nodes[threadIdx.x] = temp_node;
        
        // Ensure all threads have written to shared memory
        __syncthreads();
        
        // Write back to global memory
        d_graph_nodes[tid] = shared_nodes[threadIdx.x];
    }
}