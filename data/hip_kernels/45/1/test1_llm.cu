#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifndef _KERNEL_H
#define _KERNEL_H
typedef struct Node {
    int starting;
    int no_of_edges;
} Node;

#endif

__global__ void test1(bool* d_graph_visited, int no_of_nodes) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Check and update using a single operation to reduce memory writes
    if (tid < no_of_nodes && !d_graph_visited[tid]) {
        d_graph_visited[tid] = true;
    }
}