#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void clock_block(clock_t *d, clock_t clock_count) {
    clock_t start_clock = clock64();
    // Reuse previous computations to reduce clock reads
    clock_t clock_offset;
    do {
        clock_offset = clock64() - start_clock;
    } while (clock_offset < clock_count);

    // Avoid unnecessary memory access check
    if (d != nullptr) {
        *d = clock_offset;
    }
}