#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void colorToGray(unsigned char *input, unsigned char *output, int height, int width)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    // Precompute the fixed scale factors for each channel.
    constexpr float rScale = 0.299f;
    constexpr float gScale = 0.587f;
    constexpr float bScale = 0.114f;

    if (row < height && col < width)
    {
        int pixelIndex = row * width + col;

        // Use direct indexing to streamline operations.
        int rgbIndex = pixelIndex * 3;
        unsigned char r = input[rgbIndex];
        unsigned char g = input[rgbIndex + 1];
        unsigned char b = input[rgbIndex + 2];

        // Compute grayscale value using precomputed scale factors.
        output[pixelIndex] = static_cast<unsigned char>(r * rScale + g * gScale + b * bScale);
    }
}