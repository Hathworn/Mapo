#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16

__global__ void matrixMul(float *M, float *N, float *P, int width)
{
    // Shared memory for tile
    __shared__ float Ms[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Ns[BLOCK_SIZE][BLOCK_SIZE];

    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    float pValue = 0;

    // Loop over tiles
    for (int tileIdx = 0; tileIdx < width / BLOCK_SIZE; ++tileIdx) {
        int mRow = row;
        int mCol = tileIdx * BLOCK_SIZE + threadIdx.x;
        int nRow = tileIdx * BLOCK_SIZE + threadIdx.y;
        int nCol = col;

        // Load a tile into shared memory
        Ms[threadIdx.y][threadIdx.x] = M[mRow * width + mCol];
        Ns[threadIdx.y][threadIdx.x] = N[nRow * width + nCol];
        __syncthreads();

        // Multiply tile elements
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            pValue += Ms[threadIdx.y][k] * Ns[k][threadIdx.x];
        }
        __syncthreads();
    }

    // Write the result
    if (row < width && col < width) {
        P[row * width + col] = pValue;
    }
}