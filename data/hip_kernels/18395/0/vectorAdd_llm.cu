#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    // Calculate index; use a single variable instead of re-computing
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure that threads process indices within bounds
    if (i < numElements)
    {
        // Execute addition using memory coalescing for better performance
        C[i] = __ldg(&A[i]) + __ldg(&B[i]);
    }
}