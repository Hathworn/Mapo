#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMul_sharedMemory(float *M, float *N, float *P, int m, int j, int n)
{
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float pValue = 0.0f;

    // Use integer division and multiplication to avoid floating-point operations
    int numPhases = (j + TILE_WIDTH - 1) / TILE_WIDTH;

    for(int ph = 0; ph < numPhases; ph++)
    {
        // Load elements into shared memory if within bounds
        if(Row < m && ph * TILE_WIDTH + tx < j)
            Mds[ty][tx] = M[Row * j + ph * TILE_WIDTH + tx];
        else
            Mds[ty][tx] = 0.0f; // Avoid undefined values

        if(Col < n && ph * TILE_WIDTH + ty < j)
            Nds[ty][tx] = N[(ph * TILE_WIDTH + ty) * n + Col];
        else
            Nds[ty][tx] = 0.0f; // Avoid undefined values
        
        __syncthreads();

        // Accumulate results
        for(int k = 0; k < TILE_WIDTH; k++)
        {
            pValue += Mds[ty][k] * Nds[k][tx];
        }
        
        __syncthreads();
    }

    // Store result if within output matrix bounds
    if(Row < m && Col < n)
        P[Row * n + Col] = pValue;
}