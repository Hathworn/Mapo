#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void meanFilter(unsigned char *input, unsigned char *output, int height, int width)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure thread operates within image boundary
    if (row < height && col < width)
    {
        int pixelIndex = row * width + col;
        int pixelNum = 0;
        int tempSum = 0;

        // Utilize shared memory for faster access within blocks
        __shared__ unsigned char tile[BLOCK_SIZE + FILTER_SIZE - 1][BLOCK_SIZE + FILTER_SIZE - 1];

        // Calculate global index and shared memory index
        int tileRow = threadIdx.y + FILTER_SIZE / 2;
        int tileCol = threadIdx.x + FILTER_SIZE / 2;

        // Load data into shared memory
        tile[tileRow][tileCol] = input[row * width + col];
        __syncthreads();

        // Compute mean using shared memory
        for (int i = -FILTER_SIZE / 2; i <= FILTER_SIZE / 2; i++)
        {
            for (int j = -FILTER_SIZE / 2; j <= FILTER_SIZE / 2; j++)
            {
                int sharedRow = tileRow + j;
                int sharedCol = tileCol + i;
                
                if (sharedCol >= 0 && sharedCol < BLOCK_SIZE + FILTER_SIZE - 1 && sharedRow >= 0 && sharedRow < BLOCK_SIZE + FILTER_SIZE - 1)
                {
                    tempSum += tile[sharedRow][sharedCol];
                    pixelNum++;
                }
            }
        }

        // Assign mean value to output
        output[pixelIndex] = tempSum / pixelNum;
    }
}