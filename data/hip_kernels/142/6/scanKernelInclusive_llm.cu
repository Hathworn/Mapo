#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scanKernelInclusive(int *c, const int *a, size_t size, size_t offset)
{
    int myId = threadIdx.x + offset;

    // Boundary check within size and total offset
    if (myId < (size + offset)) 
    {
        c[myId] = a[myId];

        // Synchronize threads to ensure all elements are copied
        __syncthreads();

        // Reduce loop iterations by using warp-level shuffle for sum
        for (unsigned int _neighbor = 1; _neighbor <= (myId - offset); _neighbor <<= 1)
        {
            int op2 = 0;
            if (myId >= _neighbor + offset) 
            {
                op2 = __shfl_up_sync(0xFFFFFFFF, c[myId], _neighbor);
            }

            // Synchronize threads between load and store operations
            __syncthreads();
            
            c[myId] += op2;

            // Synchronize threads after update
            __syncthreads();
        }

        // Deal with offset for final scan result
        if (offset > 0) 
        {
            c[myId] += c[offset - 1];
        }
    }
}