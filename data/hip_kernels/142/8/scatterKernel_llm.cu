#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scatterKernel(const unsigned int *d_In, const unsigned int *d_InPos, const unsigned int *d_FalseKeyAddresses, unsigned int *d_Out, unsigned int *d_OutPos, const unsigned int totalFalses, size_t size, unsigned int bitPos)
{
    // Optimize thread calculation using flat block index
    int myId = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within bounds
    if (myId < size)
    {
        unsigned int _myFalseKeyAddress = d_FalseKeyAddresses[myId];

        // Calculate true sort key address
        int _trueSortKeyAddress = myId - _myFalseKeyAddress + totalFalses;

        // True sort key?
        unsigned int _trueSortKey = (d_In[myId] >> bitPos) & 0x1;

        // Ternary operator for destination address
        int _destinationAddress = _trueSortKey ? _trueSortKeyAddress : _myFalseKeyAddress;

        // Directly set values in output arrays
        d_Out[_destinationAddress] = d_In[myId];
        d_OutPos[_destinationAddress] = d_InPos[myId];
    }
}