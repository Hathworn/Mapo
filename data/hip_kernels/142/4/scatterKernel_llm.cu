#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scatterKernel(const unsigned int *d_In, const unsigned int *d_FalseKeyAddresses, unsigned int *d_Out, const unsigned int totalFalses, size_t size, unsigned int bitPos)
{
    // Calculate unique thread index for this execution thread
    int myId = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread operates only within data size
    if (myId < size)
    {
        // Read false key address efficiently using local variable
        unsigned int myFalseKeyAddress = d_FalseKeyAddresses[myId];

        // Calculate true sort key address directly using thread index calculations
        int trueSortKeyAddress = myId - myFalseKeyAddress + totalFalses;

        // Determine if key is true or false using shift and mask
        unsigned int trueSortKey = (d_In[myId] >> bitPos) & 0x1;

        // Utilize conditional to compute destination address
        int destinationAddress = trueSortKey ? trueSortKeyAddress : myFalseKeyAddress;

        // Write sorted data to output in reduced latency format
        d_Out[destinationAddress] = d_In[myId];
    }
}