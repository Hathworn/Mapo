#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void splitKernel(const unsigned int *d_In, unsigned int *d_Out, size_t size, unsigned int bitPos, unsigned int *lastValue)
{
    // Precompute commonly used values for efficiency
    int threadId = threadIdx.x + (threadIdx.y * blockDim.x);
    int blockId = (blockIdx.y * gridDim.x) + blockIdx.x;
    int threadsPerBlock = blockDim.x * blockDim.y;
    int myId = (blockId * threadsPerBlock) + threadId;

    if (myId < size) {
        // Simplified and optimized the ternary operation
        unsigned int _tmp = ((d_In[myId] >> bitPos) & 0x1) ^ 0x1;
        d_Out[myId] = _tmp;

        // Only set lastValue if it's provided and myId is the last element
        if ((lastValue != nullptr) && (myId == size - 1)) {
            *lastValue = _tmp;
        }
    }
}