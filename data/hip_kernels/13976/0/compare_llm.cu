#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZE 16

__global__ void compare(int *in_d, int* out_d)
{
    // Use thread ID to remove conditional branching and increase parallelism
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < SIZE) // Add boundary check to prevent out-of-bounds access
    {
        out_d[idx] = (in_d[idx] == 6) ? 1 : 0;
    }
}