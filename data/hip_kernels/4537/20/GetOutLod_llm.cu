#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod, const size_t lod_len, size_t* out_lod0) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < lod_len) {
        // Load values into registers for faster access
        size_t in_value = in_lod[index];
        size_t num_value = num_erased[in_value];
        // Perform computation and store result
        out_lod0[index] = in_value - num_value;
    }
}