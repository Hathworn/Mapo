#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_1_kernel(int h1d, int h2d, int h3d, int p4d, int p6d, int p4ld_t2, int h1ld_t2, int h3ld_v2, int h2ld_v2, int p6ld_v2, int h3ld_t3, int h2ld_t3, int h1ld_t3, int p6ld_t3, int p4ld_t3, double *t2_d, double *v2_d, int p4, int total_x, double* t3d) {
    int h1, h2, h3, p6;
    __shared__ double t2_shm[T1 * 2 * Tcomm];

    // Efficient global memory to shared memory transfer
    for (int i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        t2_shm[i] = t2_d[i];
    }
    
    __syncthreads(); // Ensure shared memory is fully populated

    int thread_x = T2 * T1 * blockIdx.x + threadIdx.x;
    int numThreads = gridDim.x * blockDim.x;
    
    // Loop unrolled over potential threads
    for (int i = thread_x; i < total_x; i += numThreads) {
        int rest_x = i;
        h3 = rest_x % h3d;
        rest_x /= h3d;
        h2 = rest_x % h2d;
        rest_x /= h2d;
        p6 = rest_x % p6d;

        // Perform computation directly with the correct index
        for (h1 = 0; h1 < h1d; h1++) {
            for (p4 = 0; p4 < p4d; p4++) {
                t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4 * p4ld_t3] += 
                    t2_shm[h1 * p4d + p4] * v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2];
            }
        }
    }
}