#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_2_kernel(int h1d, int h2d, int h3d, int p4d, int p6d, int p4ld_t2, int h1ld_t2, int h3ld_v2, int h2ld_v2, int p6ld_v2, int h3ld_t3, int h2ld_t3, int h1ld_t3, int p6ld_t3, int p4ld_t3, double *t2_d, double *v2_d, int p4, int total_x, double* t3d) {
    int h1, h2, h3, p6;
    __shared__ double t2_shm[T1 * 2 * Tcomm];

    // Efficiently load t2_d into shared memory using all threads
    for (int i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        t2_shm[i] = t2_d[i];
    }
    __syncthreads();
  
    int thread_x = blockIdx.x * blockDim.x + threadIdx.x;

    for (int base_idx = thread_x; base_idx < total_x; base_idx += gridDim.x * blockDim.x) {
        int idx = base_idx;
        
        h3 = idx % h3d;
        idx /= h3d;
        
        h2 = idx % h2d;
        idx /= h2d;
        
        p6 = idx % p6d;

        for (h1 = 0; h1 < h1d; h1++) {
            for (p4 = 0; p4 < p4d; p4++) {
                // Optimize data access by computing only required indices
                int t3d_idx = h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4 * p4ld_t3;
                int t2_shm_idx = h1 * p4d + p4;
                int v2_d_idx = h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2;

                t3d[t3d_idx] -= t2_shm[t2_shm_idx] * v2_d[v2_d_idx];
            }
        }
    }
}