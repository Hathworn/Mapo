#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_8_kernel(int h1d, int h2d, int h3d, int p4d, int p6d, int p4ld_t2, int h1ld_t2, int h3ld_v2, int h2ld_v2, int p6ld_v2, int h3ld_t3, int h2ld_t3, int h1ld_t3, int p6ld_t3, int p4ld_t3, double *t3d, double *t2_d, double *v2_d, int p4, int total_x) {
  int h1, h2, h3, p6;
  __shared__ double t2_shm[T1 * 2 * Tcomm];

  // Use a loop enclosing conditional to prevent unnecessary extra operations
  for (int i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
    t2_shm[i] = t2_d[i];
  }
  __syncthreads();
  
  // Optimize the division and modulus with simpler operations
  int rest_x = blockIdx.x;
  int thread_x = T2 * T1 * rest_x + threadIdx.x;
  
  // Move the computation into loop avoiding extraneous calculation
  for (int i = thread_x; i < total_x; i += gridDim.x * blockDim.x) {
    int temp_x = i;
    h3 = temp_x % h3d; temp_x /= h3d;
    h2 = temp_x % h2d; temp_x /= h2d;
    p6 = temp_x % p6d;

    if ((thread_x + i) < total_x) {
      for (h1 = 0; h1 < h1d; h1++) {
        for (p4 = 0; p4 < p4d; p4++) {
          // Reduce the number of memory accesses by calculating indices once
          int idx_t3 = h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4 * p4ld_t3;
          int idx_t2 = h1 * p4d + p4;
          int idx_v2 = h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2;
          t3d[idx_t3] -= t2_shm[idx_t2] * v2_d[idx_v2];
        }
      }
    }
  }
  __syncthreads();
}