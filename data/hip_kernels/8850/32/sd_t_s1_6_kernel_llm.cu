#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_6_kernel(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d, 
                                 int p4ld_t2, int h1ld_t2, int h3ld_v2, int h2ld_v2, 
                                 int p6ld_v2, int p5ld_v2, int h3ld_t3, int h2ld_t3, 
                                 int h1ld_t3, int p6ld_t3, int p5ld_t3, int p4ld_t3, 
                                 double *t3d, double *t2_d, double *v2_d, int p4, int total_x) {
    int h1, h2, h3, p6, p5;
    __shared__ double t2_shm[T1 * 2 * Tcomm];

    // Load data into shared memory efficiently using coalesced memory access
    int idx = threadIdx.x;
    while (idx < h1d * p4d) {
        t2_shm[idx] = t2_d[idx];
        idx += blockDim.x;
    }
    __syncthreads();

    // Calculate thread-specific indices for processing
    int rest_x = blockIdx.x;
    int thread_x = T2 * T1 * rest_x + threadIdx.x;
    rest_x = thread_x;

    // Begin loop for computation, optimizing with coalesced memory access
    for (int i = 0; i < total_x; i += gridDim.x * blockDim.x) {
        rest_x += i;
        h3 = rest_x % h3d;
        rest_x /= h3d;
        h2 = rest_x % h2d;
        rest_x /= h2d;
        p6 = rest_x % p6d;
        rest_x /= p6d;
        p5 = rest_x % p5d;

        if ((thread_x + i) < total_x) {
            for (h1 = 0; h1 < h1d; h1++) {
                for (p4 = 0; p4 < p4d; p4++) {
                    // Cache-aware operations to minimize latency
                    int t3_index = h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 +
                                   p6 * p6ld_t3 + p5 * p5ld_t3 + p4 * p4ld_t3;
                    int t2_index = h1 * p4d + p4;
                    int v2_index = h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2 + p5 * p5ld_v2;
                    t3d[t3_index] -= t2_shm[t2_index] * v2_d[v2_index];
                }
            }
        }
    }
    __syncthreads();
}