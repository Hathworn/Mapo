#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sd_t_s1_2_kernel(int h1d,int h2d,int h3d,int p4d,int p6d,int p4ld_t2,int h1ld_t2,int h3ld_v2,int h2ld_v2,int p6ld_v2,int h3ld_t3,int h2ld_t3,int h1ld_t3,int p6ld_t3,int p4ld_t3,double *t2_d, double *v2_d,int p4, int total_x, double* t3d) {
    int h1,h2,h3,p6;
    extern __shared__ double t2_shm[]; // Dynamically allocate shared memory

    for(int i=threadIdx.x; i<h1d*p4d; i+=blockDim.x)
        if(i<h1d*p4d)
            t2_shm[i] = t2_d[i]; // Efficiently load t2_d into shared memory

    int rest_x = blockIdx.x;
    int thread_x = T2*T1*rest_x + threadIdx.x;
    __syncthreads(); // Ensure all shared memory loads are complete

    for(int i=thread_x; i<total_x; i+=gridDim.x*blockDim.x) {
        int r = i;
        h3 = r % h3d;
        r /= h3d;
        h2 = r % h2d;
        r /= h2d;
        p6 = r % p6d;

        for(h1=0; h1<h1d; h1++) {
            for(p4=0; p4<p4d; p4++) {
                // Use registers for local computations
                double t2_element = t2_shm[h1 * p4d + p4];
                double v2_element = v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2];

                t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4 * p4ld_t3] -= t2_element * v2_element;
            }
        }
    }
    __syncthreads(); // Ensure all stores are complete before kernel exits
}