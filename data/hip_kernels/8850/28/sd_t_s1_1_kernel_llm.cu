#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_1_kernel(int h1d, int h2d, int h3d, int p4d, int p6d, int p4ld_t2, int h1ld_t2, int h3ld_v2, int h2ld_v2, int p6ld_v2, int h3ld_t3, int h2ld_t3, int h1ld_t3, int p6ld_t3, int p4ld_t3, double *t2_d, double *v2_d, int p4, int total_x, double* t3d) {
    int h1, h2, h3, p6;
    extern __shared__ double t2_shm[];

    // Load data into shared memory and ensure threads do not load redundant data
    for (int i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        t2_shm[i] = t2_d[i];
    }
    __syncthreads();

    int rest_x = blockIdx.x;
    int thread_x = T2 * T1 * rest_x + threadIdx.x;

    // Efficiently loop through data using grid-stride loop to handle large data sizes
    for (int i = thread_x; i < total_x; i += gridDim.x * blockDim.x) {
        rest_x = i;
        h3 = rest_x % h3d;
        rest_x /= h3d;
        h2 = rest_x % h2d;
        rest_x /= h2d;
        p6 = rest_x % p6d;
        
        // Optimize nested loops by removing redundant computations
        for (h1 = 0; h1 < h1d; h1++) {
            for (p4 = 0; p4 < p4d; p4++) {
                t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4 * p4ld_t3] += 
                    t2_shm[h1 * p4d + p4] * v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2];
            }
        }
    }
    __syncthreads();
}