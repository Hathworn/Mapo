#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sd_t_s1_7_kernel(int h1d,int h2d,int h3d,int p4d,int p6d,int p4ld_t2,int h1ld_t2,int h3ld_v2,int h2ld_v2,int p6ld_v2,int h3ld_t3,int h2ld_t3,int h1ld_t3,int p6ld_t3,int p4ld_t3,double *t3d, double *t2_d, double *v2_d,int p4, int total_x) {
    int h1, h2, h3, p6;
    extern __shared__ double t2_shm[]; // Dynamic shared memory allocation

    // Load t2_d into shared memory
    for(int i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        if(i < h1d * p4d) {
            t2_shm[i] = t2_d[i];
        }
    }
    __syncthreads();

    int thread_x = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_x < total_x) {
        // Calculate the initial indices
        int index = thread_x;
        h3 = index % h3d;
        index /= h3d;
        h2 = index % h2d;
        index /= h2d;
        p6 = index % p6d;

        // Iterate over tile group
        for(h1 = 0; h1 < h1d; h1++) {
            for(int p4_local = 0; p4_local < p4d; p4_local++) {
                atomicAdd(&t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4_local * p4ld_t3], 
                          t2_shm[h1 * p4d + p4_local] * v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2]);
            }
        }
    }
}