#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sd_t_s1_8_kernel(int h1d, int h2d, int h3d, int p4d, int p6d, int p4ld_t2, int h1ld_t2, int h3ld_v2, int h2ld_v2, int p6ld_v2, int h3ld_t3, int h2ld_t3, int h1ld_t3, int p6ld_t3, int p4ld_t3, double *t3d, double *t2_d, double *v2_d, int p4, int total_x) {
  int h1, h2, h3, p6;
  __shared__ double t2_shm[T1 * 2 * Tcomm];

  // Load t2_d into shared memory
  for (int i = threadIdx.x; i < h1d * p4d; i += blockDim.x)
    if (i < h1d * p4d)
      t2_shm[i] = t2_d[i];
  __syncthreads(); // Ensure all threads have loaded t2_d

  int rest_x = blockIdx.x * blockDim.x + threadIdx.x;
  
  // Loop to cover total_x
  for (int offset = rest_x; offset < total_x; offset += gridDim.x * blockDim.x) {
    int idx = offset;
    h3 = idx % h3d;
    idx /= h3d;
    h2 = idx % h2d;
    idx /= h2d;
    p6 = idx % p6d;

    // Compute inside the bounds of total_x
    if (offset < total_x) {
      double v2_val = v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2]; // Pre-compute v2 lookup
      for (h1 = 0; h1 < h1d; h1++) {
        for (p4 = 0; p4 < p4d; p4++) {
          // Update t3d with pre-computed value
          t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4 * p4ld_t3] -= t2_shm[h1 * p4d + p4] * v2_val;
        }
      }
    }
  }
  __syncthreads(); // Ensure all threads finish computation before ending kernel
}