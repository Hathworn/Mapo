#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sd_t_s1_6_kernel(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d, int p4ld_t2, int h1ld_t2, int h3ld_v2, int h2ld_v2, int p6ld_v2, int p5ld_v2, int h3ld_t3, int h2ld_t3, int h1ld_t3, int p6ld_t3, int p5ld_t3, int p4ld_t3, double *t3d, double *t2_d, double *v2_d, int p4, int total_x) {
    int h1, h2, h3, p6, p5;
    __shared__ double t2_shm[T1 * 2 * Tcomm];

    // Load data into shared memory
    for (int i = threadIdx.x; i < h1d * p4d; i += blockDim.x)
        if (i < h1d * p4d)
            t2_shm[i] = t2_d[i];
    __syncthreads(); // Ensure all data is loaded

    int rest_x = blockIdx.x;
    int thread_x = T2 * T1 * rest_x + threadIdx.x;
    for (int i = thread_x; i < total_x; i += gridDim.x * blockDim.x) {
        rest_x = i;
        h3 = rest_x % h3d;
        rest_x = rest_x / h3d;
        h2 = rest_x % h2d;
        rest_x = rest_x / h2d;
        p6 = rest_x % p6d;
        rest_x = rest_x / p6d;
        p5 = rest_x % p5d;

        if (i < total_x)
            for (h1 = 0; h1 < h1d; h1++)
                for (p4 = 0; p4 < p4d; p4++) {
                    // Update t3d with computed values
                    t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p5 * p5ld_t3 + p4 * p4ld_t3] -=
                        t2_shm[h1 * p4d + p4] * v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2 + p5 * p5ld_v2];
                }
    }
    __syncthreads(); // Ensure all threads have completed
}