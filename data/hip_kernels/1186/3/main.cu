#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "gpu_rBRIEF_naive.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float4 *workload = NULL;
hipMalloc(&workload, XSIZE*YSIZE);
int *output = NULL;
hipMalloc(&output, XSIZE*YSIZE);
int4 *pattern = NULL;
hipMalloc(&pattern, XSIZE*YSIZE);
int4 *train_bin_vec = NULL;
hipMalloc(&train_bin_vec, XSIZE*YSIZE);
int K = 1;
int P = 1;
int I = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
gpu_rBRIEF_naive<<<gridBlock, threadBlock>>>(workload,output,pattern,train_bin_vec,K,P,I);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
gpu_rBRIEF_naive<<<gridBlock, threadBlock>>>(workload,output,pattern,train_bin_vec,K,P,I);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
gpu_rBRIEF_naive<<<gridBlock, threadBlock>>>(workload,output,pattern,train_bin_vec,K,P,I);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}