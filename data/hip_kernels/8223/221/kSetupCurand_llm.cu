#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    // Avoid redundant calculations by computing the state pointer once
    hiprandState* localState = &state[tidx];

    // Initialize random generator state; each thread uses unique sequence
    hiprand_init(seed, tidx, 0, localState);
}