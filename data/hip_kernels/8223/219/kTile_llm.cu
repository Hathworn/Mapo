#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Compute unique thread index using block and thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the total number of threads
    const int numThreads = blockDim.x * gridDim.x;
    // Loop over each target element assigned to this thread
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Calculate 2D coordinate in the target matrix
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        // Map target coordinates to source coordinates using modulo
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Assign value from source to target matrix
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}