#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use blockIdx.x and threadIdx.x to differentiate threads
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        // Only one thread performs the division to prevent race conditions
        *accuracy /= N;
    }
}