#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    extern __shared__ float reduction_buffer[]; // Use dynamic shared memory
    float tmp = 0.0f;
    int idx = threadIdx.x;
    int offset = blockIdx.x * dim;
    
    // Compute local sum for reduction
    for (int i = idx; i < dim; i += blockDim.x) {
        float y_val = Y[offset + i];
        tmp += dY[offset + i] * y_val;
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Warp reduction to speed up global reduction
    if (idx < 32) {
        for (int i = blockDim.x / 2; i > 32; i >>= 1) {
            tmp += reduction_buffer[idx + i];
            reduction_buffer[idx] = tmp;
            __syncthreads();
        }
    }
    
    // Final warp-level reduction
    if (idx < 32) {
        for (int i = 16; i > 0; i >>= 1) {
            tmp += __shfl_down(tmp, i);
        }
    }

    if (idx == 0) reduction_buffer[0] = tmp;
    __syncthreads();

    // Compute gradient.
    float sum = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[offset + i] = Y[offset + i] * (dY[offset + i] - sum);
    }
}