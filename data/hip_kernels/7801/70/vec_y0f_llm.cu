#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Optimize kernel by reducing register pressure and using efficient indexing
__global__ void vec_y0f (size_t n, float *result, float *x)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        float xi = x[id]; // Load input into a register
        result[id] = y0f(xi); // Use register variable
    }
}