#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel for vector-scalar subtraction
__global__ void vec_subScalarf(size_t n, float *result, float *x, float y)
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform operation if within bounds
    if (id < n)
    {
        result[id] = x[id] - y;
    }
}