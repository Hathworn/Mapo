#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Kernel function to compute the normcdff of a vector
__global__ void vec_normcdff (size_t n, float *result, float *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure valid thread processing
    if (id < n)
    {
        result[id] = normcdff(y[id]); // Calculate normcdff
    }
}