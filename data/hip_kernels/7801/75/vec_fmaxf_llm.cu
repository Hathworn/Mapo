#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// === Vector math (two arguments) ============================================

__global__ void vec_fmaxf(size_t n, float *result, float *x, float *y) 
{
    // Compute the global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure thread processes a valid element
    if (id < n) 
    {
        // Perform the element-wise maximum operation
        result[id] = fmaxf(x[id], y[id]);
    }
}