#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

#define BLOCK_SIZE 256  // Defining block size for optimal performance

extern "C"

__global__ void vec_exp10f(size_t n, float *result, float *x)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = id; i < n; i += stride) {  // Process using grid stride loop
        result[i] = exp10f(x[i]);
    }
}