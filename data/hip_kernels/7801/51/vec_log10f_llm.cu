#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Kernel to calculate the base 10 logarithm for each element in the array.
__global__ void vec_log10f(size_t n, float *result, float *x)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Use loop unrolling for better performance
    for (int i = id; i < n; i += stride)
    {
        result[i] = log10f(x[i]);
    }
}