#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* 
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
* 
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Kernel function to calculate the arc tangent of each element in the array
__global__ void vec_atanf(size_t n, float *result, float *x) 
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Check if thread ID is within range of input array size
    if (id < n)
    {
        result[id] = atanf(x[id]); // Calculate arc tangent
    }
}