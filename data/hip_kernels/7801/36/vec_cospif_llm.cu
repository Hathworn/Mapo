#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Optimize kernel function for better performance
__global__ void vec_cospif (size_t n, float *result, float *x)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Ensure to cover all elements with a stride
    for (int i = id; i < n; i += stride) {
        result[i] = cospif(x[i]);
    }
}