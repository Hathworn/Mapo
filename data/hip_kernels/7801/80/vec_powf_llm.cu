#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Global kernel for element-wise power operation
__global__ void vec_powf(size_t n, float *result, float *x, float *y)
{
    // Use more efficient calculation of thread index
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check to ensure within bounds
    if (id < n)
    {
        result[id] = powf(x[id], y[id]);
    }
}