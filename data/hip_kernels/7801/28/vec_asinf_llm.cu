#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize vec_asinf kernel function
__global__ void vec_asinf (size_t n, float *result, float *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundary to avoid unnecessary computation
    if (id < n) {
        result[id] = asinf(x[id]);
    }
}