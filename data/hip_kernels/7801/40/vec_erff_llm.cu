#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

extern "C"

// Improved vector and scalar arithmetic kernel 
extern "C"

//=== Vector math (one argument) =============================================

// Calculate the error function of the input argument.
__global__ void vec_erff(size_t n, float* result, float* x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Use consistent thread ID calculation
    if (id < n) {
        result[id] = erff(x[id]); // Efficient memory access and computation
    }
}