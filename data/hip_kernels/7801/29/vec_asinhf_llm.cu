#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

extern "C"

//=== Vector arithmetic ======================================================

extern "C"

//=== Vector-and-scalar arithmetic ===========================================

extern "C"

//=== Vector comparison ======================================================

extern "C"

//=== Vector-and-scalar comparison ===========================================

extern "C"

//=== Vector math (one argument) =============================================

extern "C"

// Calculate the arc hyperbolic sine of the input argument.
__global__ void vec_asinhf(size_t n, float *result, float *x)
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds
    if (id < n)
    {
        // Compute arc hyperbolic sine for each element
        result[id] = asinhf(x[id]);
    }
}