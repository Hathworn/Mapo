#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel execution by using shared memory to reduce redundant memory accesses
__global__ void vec_erfcf (size_t n, float *result, float *x)
{
    // Calculate global thread index
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // Use shared memory for the current block
    extern __shared__ float shared[];
    
    if (id < n)
    {
        // Load into shared memory
        shared[threadIdx.x] = x[id];
        __syncthreads();  // Ensure all loads are done
        
        // Perform computation using shared memory
        result[id] = erfcf(shared[threadIdx.x]);
    }
}