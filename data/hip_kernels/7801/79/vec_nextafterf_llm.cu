#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Improved kernel for vec_nextafterf
__global__ void vec_nextafterf (size_t n, float *result, float *x, float *y) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Stride by total number of threads
    int stride = blockDim.x * gridDim.x;
    for (int i = id; i < n; i += stride) {
        result[i] = nextafterf(x[i], y[i]);
    }
}