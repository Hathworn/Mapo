#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

extern "C"

//=== Vector arithmetic ======================================================

extern "C"

//=== Vector-and-scalar arithmetic ===========================================

extern "C"

//=== Vector comparison ======================================================

extern "C"

//=== Vector-and-scalar comparison ===========================================

extern "C"

//=== Vector math (one argument) =============================================

extern "C"

//=== Vector math (two arguments) ============================================

extern "C"

__global__ void vec_mulf (size_t n, float *result, float  *x, float  *y)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we have a complete warp to optimize memory access
    if (id < n)
    {
        result[id] = x[id] * y[id];
    }
}