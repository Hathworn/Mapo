#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Optimized kernel using grid-stride loop for improved parallelism
__global__ void vec_addScalarf(size_t n, float *result, float *x, float y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while (id < n)
    {
        result[id] = x[id] + y;
        id += stride;
    }
}