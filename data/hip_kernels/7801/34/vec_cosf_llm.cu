#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

extern "C"


// Calculate the cosine of the input argument.
extern "C"

__global__ void vec_cosf(size_t n, float *result, float *x)
{
    // Calculate global thread ID
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Vectorize loads for better memory access pattern
    int stride = blockDim.x * gridDim.x;
    for (int i = id; i < n; i += stride)
    {
        // Calculate cosine
        result[i] = cosf(x[i]);
    }
}