#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Optimize memory access and reduce divergence
__global__ void vec_nef (size_t n, float *result, float  *x, float  *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        float xi = x[id];
        float yi = y[id];
        result[id] = (xi != yi) ? 1.0f : 0.0f;
    }
}