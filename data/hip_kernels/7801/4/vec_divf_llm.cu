#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Optimize vector division kernel using shared memory for coalesced access
__global__ void vec_divf(size_t n, float* result, float* x, float* y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // Use shared memory for better memory access pattern
    extern __shared__ float shared[];
    if (id < n)
    {
        shared[threadIdx.x] = y[id];
        __syncthreads(); // Ensure all threads have copied data to shared memory
        result[id] = x[id] / shared[threadIdx.x];
    }
}