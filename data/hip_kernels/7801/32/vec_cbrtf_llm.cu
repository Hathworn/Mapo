#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

extern "C"

// Optimize global kernel function using appropriate memory and thread strategy

__global__ void vec_cbrtf(size_t n, float *result, float *x)
{
    // Calculate unique thread index
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Loop optimization for improving memory access
    for (int index = id; index < n; index += gridDim.x * blockDim.x) {
        result[index] = cbrtf(x[index]);
    }
}