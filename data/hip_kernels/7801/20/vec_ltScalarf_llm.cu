#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Optimized kernel to check if elements in vector x are less than scalar y
__global__ void vec_ltScalarf(size_t n, float *result, float *x, float y)
{
    // Calculate the global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check boundary condition and perform the operation
    if (id < n)
    {
        result[id] = (x[id] < y) ? 1.0f : 0.0f;
    }
}