#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_eqf (size_t n, float *result, float *x, float *y)
{
    // Calculate the global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Shared memory for better data locality (optional, depends on context)
    // __shared__ float shm_result[blockDim.x];
    // __shared__ float shm_x[blockDim.x];
    // __shared__ float shm_y[blockDim.x];

    // Check bounds and perform the computation
    if (id < n) 
    {
        result[id] = (x[id] == y[id]) ? 1.0f : 0.0f;
    }
}