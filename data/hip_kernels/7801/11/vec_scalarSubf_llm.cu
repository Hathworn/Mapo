#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//=== Vector arithmetic ======================================================
//=== Vector-and-scalar arithmetic ===========================================
//=== Vector comparison ======================================================
//=== Vector-and-scalar comparison ===========================================
//=== Vector math (one argument) =============================================
//=== Vector math (two arguments) ============================================

__global__ void vec_scalarSubf(size_t n, float *result, float x, float *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        // Optimized subtraction operation, directly accessed
        result[id] = x - y[id];
    }
}