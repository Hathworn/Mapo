#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 256  // Optimal block size for improved performance

__global__ void vec_floorf (size_t n, float *result, float  *x)
{
    int id = threadIdx.x + blockIdx.x * BLOCK_SIZE;
    int stride = BLOCK_SIZE * gridDim.x;

    // Loop over elements with stride for better utilization
    for (int i = id; i < n; i += stride) 
    {
        result[i] = floorf(x[i]);
    }
}