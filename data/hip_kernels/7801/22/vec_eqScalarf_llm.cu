#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

extern "C"


__global__ void vec_eqScalarf (size_t n, float *result, float *x, float y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Unroll loop for efficiency
    int stride = blockDim.x * gridDim.x;
    for (int i = id; i < n; i += stride)
    {
        result[i] = (x[i] == y) ? 1.0f : 0.0f;
    }
}