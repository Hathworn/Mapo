#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_rcbrtf(size_t n, float *result, float *x)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure id is within bounds
    if (id < n)
    {
        // Calculate reciprocal cube root
        result[id] = rcbrtf(x[id]);
    }
}