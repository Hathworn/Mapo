#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

extern "C"

__global__ void vec_hypotf(size_t n, float *result, float *x, float *y)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Use grid-stride loop for better utilization
    for (int i = id; i < n; i += blockDim.x * gridDim.x)
    {
        result[i] = hypotf(x[i], y[i]);
    }
}