#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//=== Vector math (one argument) =============================================

__global__ void vec_erfcxf(size_t n, float *result, float *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        // Use shared memory to reduce global memory access
        __shared__ float shared_x[256]; // Ensure the size fits within blockDim.x
        shared_x[threadIdx.x] = x[id];
        __syncthreads();

        // Compute result using shared memory and store back into global memory
        result[id] = erfcxf(shared_x[threadIdx.x]);
    }
}