#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

__global__ void vec_setf(size_t n, float *result, float value)
{
    // Calculate unique thread index
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use a loop with stride equal to total number of threads
    for (int i = id; i < n; i += blockDim.x * gridDim.x) 
    {
        result[i] = value;
    }
}