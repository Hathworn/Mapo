#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

extern "C"

// Calculate the arc cosine of the input argument.
extern "C"

// Optimize using loop unrolling
__global__ void vec_acosf (size_t n, float *result, float *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Unrolling the loop to improve performance
    int stride = gridDim.x * blockDim.x;
    while (id < n)
    {
        result[id] = acosf(x[id]);
        
        // Process next element in the stride
        if (id + stride < n)
            result[id + stride] = acosf(x[id + stride]);

        // Increment index by stride for next iteration
        id += stride;
    }
}