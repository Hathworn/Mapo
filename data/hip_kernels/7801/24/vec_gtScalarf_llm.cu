#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_gtScalarf (size_t n, float *result, float *x, float y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Prefetch the value into a register to reduce memory access
    if (id < n)
    {
        float x_val = x[id];
        result[id] = (x_val > y) ? 1.0f : 0.0f;
    }
}