#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/


//=== Vector math (one argument) =============================================

// Calculate the inverse complementary error function of the input argument.
extern "C"

__global__ void vec_erfcinvf(size_t n, float *result, float *y)
{
    // Calculate flattened global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use bounds checking to prevent out-of-range memory access
    if (id < n)
    {
        // Compute the inverse complementary error function
        result[id] = erfcinvf(y[id]);
    }
}