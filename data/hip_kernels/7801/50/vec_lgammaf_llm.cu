#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Optimize kernel using unrolling and efficient memory access
__global__ void vec_lgammaf(size_t n, float *result, float *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = id; i < n; i += stride) // Ensure all threads are utilized
    {
        result[i] = lgammaf(x[i]);
    }
}