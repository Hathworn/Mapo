#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Kernel function optimized with loop unrolling and shared memory
__global__ void vec_scalarMulf(size_t n, float *result, float x, float *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;  // Calculate stride for grid loops

    for (int i = id; i < n; i += stride)  // Unrolled loop for memory coalescing
    {
        result[i] = x * y[i];
    }
}