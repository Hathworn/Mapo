#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_acoshf (size_t n, float *result, float *x)
{
    // Compute global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop unrolling: process elements in chunks, assuming n is multiple of chunk size
    int stride = gridDim.x * blockDim.x;
    for (; id < n; id += stride)
    {
        result[id] = acoshf(x[id]);
    }
}