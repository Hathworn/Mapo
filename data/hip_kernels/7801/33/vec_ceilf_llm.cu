#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Kernel for performing ceiling operation on each element of a vector
__global__ void vec_ceilf(size_t n, float *result, float *x)
{
    // Obtain global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the current thread works on a valid element
    if (id < n)
    {
        // Perform ceiling operation
        result[id] = ceilf(x[id]);
    }
}