#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Optimize kernel using shared memory and loop unrolling to improve performance

__global__ void vec_tgammaf(size_t n, float *result, float *x) 
{
    extern __shared__ float shmem[];
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int id = threadIdx.x;

    if (idx < n) {
        // Load data into shared memory for better memory access patterns
        shmem[id] = x[idx];
        __syncthreads();

        // Unroll loop for better instruction throughput
        #pragma unroll
        for (int i = id; i < blockDim.x && idx+i < n; i += blockDim.x) {
            result[idx+i] = tgammaf(shmem[i]);
        }
    }
}