#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

extern "C"


// Optimized kernel function for vector addition
__global__ void vec_addf (size_t n, float *result, float  *x, float  *y)
{
    // Optimize memory access patterns
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = id; i < n; i += stride)
    {
        result[i] = x[i] + y[i];
    }
}