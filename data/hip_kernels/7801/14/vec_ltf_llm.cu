#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

extern "C"

__global__ void vec_ltf (size_t n, float *result, float  *x, float  *y)
{
    // Calculate the global thread ID
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimize condition check and ensure all threads are utilized
    if (id < n)
    {
        // Simplify comparison using ternary operator
        result[id] = (x[id] < y[id]) ? 1.0f : 0.0f;
    }
}