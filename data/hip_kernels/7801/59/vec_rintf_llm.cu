#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Optimize kernel using shared memory and loop unrolling
__global__ void vec_rintf(size_t n, float *result, float *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // Use shared memory to reduce global memory accesses
    __shared__ float shared_x[256];
    
    if (id < n)
    {
        // Load data into shared memory
        shared_x[threadIdx.x] = x[id];
        __syncthreads();
        
        // Unroll loop for optimization
        #pragma unroll
        for (int i = 0; i < 1; ++i) 
        {
            result[id] = rintf(shared_x[threadIdx.x]);
        }
    }
}