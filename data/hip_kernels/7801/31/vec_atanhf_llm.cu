#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Optimized kernel using dynamic parallelism and shared memory
extern "C"
__global__ void vec_atanhf(size_t n, float *result, float *x) {
    __shared__ float sharedData[256]; // Allocate shared memory for a block

    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (id < n) {
        sharedData[threadIdx.x] = x[id];
    }
    __syncthreads(); // Synchronize threads

    // Perform computation using shared memory
    if (id < n) {
        result[id] = atanhf(sharedData[threadIdx.x]);
    }
}