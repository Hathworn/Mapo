#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

extern "C"

// Optimized kernel for computing the inverse error function
__global__ void vec_erfinvf(size_t n, float *result, float *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        // Utilize registers efficiently for computation
        float value = y[id];
        result[id] = erfinvf(value);
    }
}
