#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//=== Vector math (two arguments) ============================================

__global__ void vec_fmodf(size_t n, float *result, float *x, float *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Improved with loop unrolling for better performance
    for (; id < n; id += gridDim.x * blockDim.x)
    {
        result[id] = fmodf(x[id], y[id]);
    }
}