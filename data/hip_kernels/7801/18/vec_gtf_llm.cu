#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

//=== Vector comparison ======================================================

// Kernel function optimized for better performance
__global__ void vec_gtf(size_t n, float *result, float *x, float *y)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory for block-wide computation caching
    __shared__ float shared_x[256];
    __shared__ float shared_y[256];

    if (id < n) {
        // Load data into shared memory
        shared_x[threadIdx.x] = x[id];
        shared_y[threadIdx.x] = y[id];
    }
    __syncthreads();

    if (id < n) {
        // Perform computation using shared memory
        result[id] = (shared_x[threadIdx.x] > shared_y[threadIdx.x]) ? 1.0f : 0.0f;
    }
}