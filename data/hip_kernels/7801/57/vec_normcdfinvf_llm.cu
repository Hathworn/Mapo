#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Optimized kernel using shared memory for better memory access patterns
__global__ void vec_normcdfinvf(size_t n, float *result, float *y)
{
    extern __shared__ float shared_y[];
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;

    // Load data into shared memory for faster access
    if (id < n)
    {
        shared_y[tid] = y[id];
        __syncthreads();

        // Calculate only when id corresponds to n and data is loaded
        if (id < n)
        {
            result[id] = normcdfinvf(shared_y[tid]);
        }
    }
}