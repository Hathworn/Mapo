#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimized for memory coalescing
__global__ void vec_copysignf(size_t n, float *result, float *x, float *y) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Use for loop to ensure memory access within bounds for additional threads
    for (int i = id; i < n; i += blockDim.x * gridDim.x) {
        result[i] = copysignf(x[i], y[i]);
    }
}