#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void vec_gtef(size_t n, float *result, float *x, float *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Use loop unrolling to handle large arrays efficiently
    for (int i = id; i < n; i += stride)
    {
        result[i] = (x[i] >= y[i]) ? 1.0f : 0.0f;
    }
}