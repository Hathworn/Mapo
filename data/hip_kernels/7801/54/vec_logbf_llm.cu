#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized vector logbf function
__global__ void vec_logbf(size_t n, float *result, float *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for loop unrolling
    for (int i = id; i < n; i += stride)
    {
        result[i] = logbf(x[i]); // Process elements in a loop with stride
    }
}