#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//=== Vector math (one argument) =============================================

__global__ void vec_coshf(size_t n, float *result, float *x) {
    // Using strided loop for better occupancy
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = id; i < n; i += stride) {
        // Replace coshf with __cosf for potential performance boost
        result[i] = __coshf(x[i]);
    }
}