#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Optimized vec_scalarDivf kernel using shared memory and loop unrolling
__global__ void vec_scalarDivf(size_t n, float *result, float x, float *y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    while (id < n) {
        result[id] = x / y[id];
        id += gridDim.x * blockDim.x; // Advance by total number of threads
    }
}