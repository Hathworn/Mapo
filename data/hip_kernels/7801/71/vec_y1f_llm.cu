#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Kernel to calculate the Bessel function of the second kind for given input `x` and store in `result`
__global__ void vec_y1f(size_t n, float *result, float *x)
{
    // Calculate unique thread ID
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Process only valid indices
    if (id < n)
    {
        // Compute Bessel function for the current input
        result[id] = y1f(x[id]);
    }
}