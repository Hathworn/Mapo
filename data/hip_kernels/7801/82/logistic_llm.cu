#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// 1 / (1 + e^(-x))
extern "C" __global__ void logistic(size_t n, double *result, double *x)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for common operation
    __shared__ double shared_exp[256];  // Adjust size based on blockDim

    if (i < n)
    {
        shared_exp[threadIdx.x] = exp(-x[i]);
        __syncthreads();  // Ensure all threads have completed the above operation

        result[i] = 1.0 / (1.0 + shared_exp[threadIdx.x]);
    }
}