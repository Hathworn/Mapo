#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Kernel optimized for coalesced memory access
__global__ void vec_truncf(size_t n, float *result, float *x)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop for better utilization
    for (int i = id; i < n; i += blockDim.x * gridDim.x) 
    {
        result[i] = truncf(x[i]);
    }
}