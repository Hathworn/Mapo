#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

extern "C"

//=== Vector arithmetic ======================================================

extern "C"


extern "C"


extern "C"


extern "C"

extern "C"




//=== Vector-and-scalar arithmetic ===========================================

extern "C"


extern "C"


extern "C"


extern "C"




extern "C"


extern "C"


extern "C"


extern "C"











//=== Vector comparison ======================================================

extern "C"


extern "C"


extern "C"


extern "C"


extern "C"



extern "C"




//=== Vector-and-scalar comparison ===========================================

extern "C"


extern "C"


extern "C"


extern "C"


extern "C"


extern "C"











//=== Vector math (one argument) =============================================


// Calculate the arc cosine of the input argument.
extern "C"


// Calculate the nonnegative arc hyperbolic cosine of the input argument.
extern "C"


// Calculate the arc sine of the input argument.
extern "C'


// Calculate the arc hyperbolic sine of the input argument.
extern "C'


// Calculate the arc tangent of the input argument.
extern "C'


// Calculate the arc hyperbolic tangent of the input argument.
extern "C'


// Calculate the cube root of the input argument.
extern "C'


// Calculate ceiling of the input argument.
extern "C'


// Calculate the cosine of the input argument.
extern "C'


// Calculate the hyperbolic cosine of the input argument.
extern "C'


// Calculate the cosine of the input argument × p .
extern "C'


// Calculate the complementary error function of the input argument.
extern "C'


// Calculate the inverse complementary error function of the input argument.
extern "C'


// Calculate the scaled complementary error function of the input argument.
extern "C'


// Calculate the error function of the input argument.
extern "C'


// Calculate the inverse error function of the input argument.
extern "C'


// Calculate the base 10 exponential of the input argument.
extern "C'


// Calculate the base 2 exponential of the input argument.
extern "C'


// Calculate the base e exponential of the input argument.
extern "C'


// Calculate the base e exponential of the input argument, minus 1.
extern "C'


// Calculate the absolute value of its argument.
extern "C'


// Calculate the largest integer less than or equal to x.
extern "C'


// Calculate the value of the Bessel function of the first kind of order 0 for the input argument.
extern "C'


// Calculate the value of the Bessel function of the first kind of order 1 for the input argument.
extern "C'


// Calculate the natural logarithm of the absolute value of the gamma function of the input argument.
extern "C'


// Calculate the base 10 logarithm of the input argument.
extern "C'


// Calculate the value of l o g e ( 1 + x ) .
extern "C'


// Calculate the base 2 logarithm of the input argument.
extern "C'


// Calculate the floating point representation of the exponent of the input argument.
extern "C'


// Calculate the natural logarithm of the input argument.
extern "C'


// Calculate the standard normal cumulative distribution function.
extern "C'


// Calculate the inverse of the standard normal cumulative distribution function.
extern "C'


// Calculate reciprocal cube root function.
extern "C'


// Round input to nearest integer value in floating-point.
extern "C'


// Round to nearest integer value in floating-point.
extern "C'


// Calculate the reciprocal of the square root of the input argument.
extern "C'


// Calculate the sine of the input argument.
extern "C'


// Calculate the hyperbolic sine of the input argument.
extern "C'


// Calculate the sine of the input argument × p .
extern "C'


// Calculate the square root of the input argument.
extern "C'


// Calculate the tangent of the input argument.
extern "C'


// Calculate the hyperbolic tangent of the input argument.
extern "C'


// Calculate the gamma function of the input argument.
extern "C'


// Truncate input argument to the integral part.
extern "C'


// Calculate the value of the Bessel function of the second kind of order 0 for the input argument.
extern "C'


// Calculate the value of the Bessel function of the second kind of order 1 for the input argument.
extern "C'









extern "C"


//--- Optimized global kernel
__global__ void vec_j1f (size_t n, float *result, float *x)
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop to handle arbitrary problem sizes
    for (int i = id; i < n; i += blockDim.x * gridDim.x)
    {
        result[i] = j1f(x[i]);
    }
}