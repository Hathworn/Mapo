#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to compute the remainder of floating point division
__global__ void vec_remainderf(size_t n, float *result, const float *x, const float *y)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure each thread processes only valid elements
    if (id < n)
    {
        result[id] = remainderf(x[id], y[id]); // Efficient single operation per thread
    }
}