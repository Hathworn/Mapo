#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for comparing each element of vector x with scalar y
__global__ void vec_gteScalarf(size_t n, float *result, float *x, float y) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Use strided access to improve global memory usage
    for (int i = id; i < n; i += blockDim.x * gridDim.x) {
        result[i] = (x[i] >= y) ? 1.0f : 0.0f;
    }
}