#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Optimize memory access by using shared memory
__global__ void vec_sqrtf(size_t n, float *result, float *x)
{
    // Calculate global index
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Load input into shared memory for coalesced access
    __shared__ float shared_x[256]; // Adjust size according to blockDim.x
    if (threadIdx.x < n - blockIdx.x * blockDim.x) {
        shared_x[threadIdx.x] = x[id];
        __syncthreads(); // Ensure all threads have loaded their data
    }
    
    // Perform square root calculation
    if (id < n) {
        result[id] = sqrtf(shared_x[threadIdx.x]);
    }
}