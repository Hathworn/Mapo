#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Negate elements of vector
__global__ void vec_negatef(size_t n, float *result, const float *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // Use grid-stride loop for better utilization
    for (int i = id; i < n; i += blockDim.x * gridDim.x)
    {
        result[i] = -x[i];
    }
}