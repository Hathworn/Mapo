#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* JCudaVec - Vector operations for JCuda
* http://www.jcuda.org
*
* Copyright (c) 2013-2015 Marco Hutter - http://www.jcuda.org
*/

// Optimize vec_lteScalarf kernel function
__global__ void vec_lteScalarf (size_t n, float *result, float *x, float y)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Use conditional assignment directly without branch
    if (id < n)
    {
        result[id] = static_cast<float>(x[id] <= y);
    }
}