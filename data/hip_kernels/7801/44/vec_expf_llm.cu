#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_expf(size_t n, float *result, float *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = __expf(x[id]); // Use fast intrinsic for expf to optimize performance
    }
}