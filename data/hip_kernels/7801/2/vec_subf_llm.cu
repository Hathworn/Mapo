#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_subf (size_t n, float *result, float *x, float *y)
{
    // Compute global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a loop to ensure all elements are processed even if grid size < n
    for (int i = id; i < n; i += blockDim.x * gridDim.x) {
        result[i] = x[i] - y[i];
    }
}