#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop with stride equal to total number of threads
    for (unsigned int i = idx; i < numEls; i += blockDim.x * gridDim.x) {
        // Perform element-wise addition
        dest[i] = a[i] + b[i];
    }
}