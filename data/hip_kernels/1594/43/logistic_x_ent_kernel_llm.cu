#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation

    if (i < n) {
        float t = truth[i];
        float p = pred[i];
        
        // Calculate terms to reduce repetitive computation
        float log_p = log(p + 1e-7);
        float log_1mp = log(1 - p + 1e-7);

        error[i] = -t * log_p - (1 - t) * log_1mp; // Calculate error
        delta[i] = t - p; // Calculate delta
    }
}