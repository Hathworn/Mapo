#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void wgan_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop over all elements with stride
    for (; i < n; i += stride) {
        // Compute error and delta
        error[i] = truth[i] ? -pred[i] : pred[i];
        delta[i] = (truth[i] > 0) ? 1.0f : -1.0f;
    }
}