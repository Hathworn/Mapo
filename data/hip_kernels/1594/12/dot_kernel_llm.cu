#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;
    
    // Loop over work items in steps of grid size
    for(int ind = index; ind < batch * n * size; ind += gridSize) {
        int f1 = ind / (size * n);
        int remainder = ind % (size * n);
        int f2 = remainder / size;
        int i = remainder % size;
        
        if (f2 <= f1) continue;

        float sum = 0;
        float norm1 = 0;
        float norm2 = 0;

        for(int b = 0; b < batch; ++b){
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            sum += output[i1] * output[i2];
            norm1 += output[i1] * output[i1];
            norm2 += output[i2] * output[i2];
        }
        
        norm1 = sqrt(norm1);
        norm2 = sqrt(norm2);
        float norm = norm1 * norm2;
        sum = sum / norm;

        for(int b = 0; b < batch; ++b){
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            atomicAdd(&delta[i1], -scale * sum * output[i2] / norm);
            atomicAdd(&delta[i2], -scale * sum * output[i1] / norm);
        }
    }
}