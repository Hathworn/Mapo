#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Loop over the elements with a stride of total threads
    for (int i = index; i < N; i += stride) {
        int in_s = i % spatial;
        int temp = i / spatial;
        int in_c = temp % layers;
        int b = temp / layers;

        int i1 = b * layers * spatial + in_c * spatial + in_s;
        int i2 = b * layers * spatial + in_s * layers + in_c;

        if (forward) out[i2] = x[i1];
        else out[i1] = x[i2];
    }
}