#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID
    int stride = gridDim.x * blockDim.x; // Calculate stride for grid
    for (int i = id; i < size; i += stride) { // Loop with stride for data parallelism
        input[i] = (rand[i] < prob) ? 0 : input[i] * scale; // Apply transformation
    }
}