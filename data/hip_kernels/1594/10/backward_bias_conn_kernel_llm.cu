#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) return;

    // Initialize shared memory for thread group reduction
    extern __shared__ float shared_sum[];
    shared_sum[threadIdx.x] = 0;

    // Accumulate delta within the batch using shared memory
    int b;
    for(b = 0; b < batch; ++b){
        int i = b * n + index;
        shared_sum[threadIdx.x] += delta[i];
    }

    // Ensure all threads have written to shared memory
    __syncthreads();

    // Use parallel reduction to sum values in shared memory
    for(int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write the result from the first thread in the block
    if (threadIdx.x == 0) {
        bias_updates[index] += shared_sum[0];
    }
}