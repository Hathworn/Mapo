#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for processing large arrays
    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x) {
        if (mask[idx] == mask_num) {
            x[idx] = val;
        }
    }
}