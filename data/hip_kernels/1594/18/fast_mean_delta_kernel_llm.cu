#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[BLOCK];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    // Initialize shared memory
    float temp_sum = 0.0f;

    // Unroll the loop for better performance
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {  // Change loop increment to `threads`
            int index = j * spatial * filters + filter * spatial + i;
            temp_sum += (i < spatial) ? delta[index] : 0.0f;
        }
    }

    // Store partial sum in shared memory
    local[id] = temp_sum;
    __syncthreads();

    // Perform reduction in shared memory
    if (id == 0) {
        mean_delta[filter] = 0.0f;
        for (int i = 0; i < threads; ++i) {
            mean_delta[filter] += local[i];
        }
        mean_delta[filter] *= (-1.0f / sqrtf(variance[filter] + 0.00001f));
    }
}