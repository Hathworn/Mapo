#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread operates within bounds
    if (i >= filters) return;
    
    float mean_sum = 0; // Use register for summation
    
    // Calculate offset for batches and spatial dimensions
    int filter_offset = i * spatial;
    
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + filter_offset + k;
            mean_sum += delta[index];
        }
    }
    
    // Normalize and negate mean_sum and store result
    mean_delta[i] = mean_sum * (-1.f / sqrtf(variance[i] + .00001f)); 
}