#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Unroll the loop to reduce overhead and increase efficiency
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; i += BLOCK*4) {
            int index0 = p + i + size*(filter + n*b);
            int index1 = index0 + BLOCK;
            int index2 = index1 + BLOCK;
            int index3 = index2 + BLOCK;
            sum += (p+i < size) ? delta[index0] : 0;
            sum += (p+i+BLOCK < size) ? delta[index1] : 0;
            sum += (p+i+BLOCK*2 < size) ? delta[index2] : 0;
            sum += (p+i+BLOCK*3 < size) ? delta[index3] : 0;
        }
    }
    part[p] = sum;
    __syncthreads();

    // Use atomic addition to avoid race conditions
    if (p == 0) {
        for (int i = 0; i < BLOCK; ++i) {
            atomicAdd(&(bias_updates[filter]), part[i]);
        }
    }
}