#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Calculate the scale outside the loop
    float scale = 1.f / (batch * spatial);
    int filter_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Exit if thread is out of bounds
    if (filter_id >= filters) return;
    
    float sum = 0.0f;
    
    // Unroll the loops for potential performance gain
    for (int j = 0; j < batch; ++j){
        for (int k = 0; k < spatial; ++k){
            int index = j * filters * spatial + filter_id * spatial + k;
            sum += x[index];
        }
    }
    
    // Apply scaling at the end
    mean[filter_id] = sum * scale;
}