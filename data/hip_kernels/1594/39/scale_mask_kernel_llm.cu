#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize boundary condition and mask check
    if (i < n && __ldg(&mask[i]) == mask_num) {
        // Scale the element if mask matches mask_num
        x[i] *= scale;
    }
}