#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index using block and grid dimensions
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    
    // Ensure thread accesses only valid indices
    for(int index = i; index < N; index += blockDim.x * gridDim.x) {
        // Apply constraints to X values at the computed index
        X[index*INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[index*INCX])); 
    }
}