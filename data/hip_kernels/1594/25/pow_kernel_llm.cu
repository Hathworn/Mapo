#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a single loop for better efficiency
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        Y[idx * INCY] = pow(X[idx * INCX], ALPHA);
    }
}