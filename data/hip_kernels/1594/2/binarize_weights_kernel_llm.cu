#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;

    extern __shared__ float shared_mean[]; // Use shared memory for mean
    int tid = threadIdx.x;
    float local_mean = 0;

    // Each thread computes partial mean
    for (int i = tid; i < size; i += blockDim.x) {
        local_mean += fabsf(weights[f * size + i]);
    }

    shared_mean[tid] = local_mean;
    __syncthreads();

    // Parallel reduction to compute full mean
    if (tid == 0) {
        float total_mean = 0;
        for (int i = 0; i < blockDim.x; ++i) {
            total_mean += shared_mean[i];
        }
        mean = total_mean / size;
        shared_mean[0] = total_mean / size;
    }
    __syncthreads();

    mean = shared_mean[0]; // All threads get the computed mean

    // Binarize weights
    for (int i = tid; i < size; i += blockDim.x) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    }
}