#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCK 512

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Unroll the first loop for better performance
    for(int b = 0; b < batch; ++b) {
        for(int i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index] * x_norm[index];
        }
    }

    part[p] = sum;
    __syncthreads();

    // Parallel reduction to accumulate shared memory results
    if (p < 256) part[p] += part[p + 256];
    __syncthreads();
    if (p < 128) part[p] += part[p + 128];
    __syncthreads();
    if (p < 64) part[p] += part[p + 64];
    __syncthreads();
    if (p < 32) {
        part[p] += part[p + 32];
        part[p] += part[p + 16];
        part[p] += part[p + 8];
        part[p] += part[p + 4];
        part[p] += part[p + 2];
        part[p] += part[p + 1];
    }

    // Update scale with the reduced sum
    if (p == 0) {
        atomicAdd(&scale_updates[filter], part[0]);  // Atomic addition to prevent race conditions
    }
}