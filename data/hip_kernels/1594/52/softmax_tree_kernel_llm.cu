#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize softmax_device function to increase parallel execution and reduce loops
__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    extern __shared__ float shared[];
    float *shared_max = shared;
    float *shared_sum = shared + blockDim.x;
    
    int i;
    float sum = 0;
    float largest = -INFINITY;

    // Parallel reduction for maximum value
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        float val = input[i * stride];
        atomicMax(&largest, val);
    }
    shared_max[threadIdx.x] = largest;
    __syncthreads();

    for (i = 1; i < blockDim.x; i *= 2) {
        int index = 2 * i * threadIdx.x;
        if (index < blockDim.x) {
            shared_max[index] = max(shared_max[index], shared_max[index + i]);
        }
        __syncthreads();
    }
    
    largest = shared_max[0];
    
    // Parallel calculation of exponential values and sum
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        float e = expf(input[i * stride] / temp - largest / temp);
        sum += e;
        output[i * stride] = e;
    }
    shared_sum[threadIdx.x] = sum;
    __syncthreads();

    for (i = 1; i < blockDim.x; i *= 2) {
        int index = 2 * i * threadIdx.x;
        if (index < blockDim.x) {
            shared_sum[index] += shared_sum[index + i];
        }
        __syncthreads();
    }
    
    sum = shared_sum[0];
    
    // Normalize the output
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        output[i * stride] /= sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;
    
    int s = id % spatial;
    id = id / spatial;
    int g = id % groups;
    int b = id / groups;
    
    int goff = group_offset[g] * spatial;
    int boff = b * stride;
    
    extern __shared__ float shared[];
    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}