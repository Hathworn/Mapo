#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global thread index

    if (index < n * size * batch) { // Ensure the thread is within bounds
        int i = index % size;
        int j = (index / size) % n; // Combine modulo and division for clarity
        int k = index / (size * n); // Simplified division

        output[(k * n + j) * size + i] += biases[j]; // Efficiently access biases
    }
}