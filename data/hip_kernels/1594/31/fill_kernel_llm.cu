#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate thread's global index
    int stride = blockDim.x * gridDim.x; // Calculate stride for grid-stride loop
    for (int i = idx; i < N; i += stride) // Use grid-stride loop for better load balancing
    {
        X[i * INCX] = ALPHA; // Set value with stride
    }
}