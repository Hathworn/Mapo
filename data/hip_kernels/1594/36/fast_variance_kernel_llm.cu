#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[BLOCK];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    float sum = 0.0f;  // Use a register for accumulation

    int total_iters = spatial * batch;
    for (int idx = id; idx < total_iters; idx += threads) {
        int batch_idx = idx / spatial;
        int spatial_idx = idx % spatial;
        int index = batch_idx * spatial * filters + filter * spatial + spatial_idx;
        
        sum += powf((x[index] - mean[filter]), 2);  // Accumulate local sum directly
    }

    local[id] = sum;  // Store accumulated sum in shared memory
    __syncthreads();

    // Use parallel reduction to accumulate shared memory results
    if (id == 0) {
        variance[filter] = 0.0f;
        for (int i = 0; i < threads; ++i) {
            variance[filter] += local[i];
        }
        variance[filter] /= (spatial * batch - 1);  // Final division
    }
}