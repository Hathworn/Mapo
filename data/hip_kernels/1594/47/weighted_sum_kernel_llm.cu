#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Use grid stride loop to handle large data sizes
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        // Compute result using weighted sum formula
        c[i] = s[i] * a[i] + (1 - s[i]) * (b ? b[i] : 0);
    }
}