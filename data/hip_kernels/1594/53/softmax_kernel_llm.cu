#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    float sum = 0.0f;
    float largest = -INFINITY;

    // Find the largest element within the input
    for(int i = 0; i < n; ++i){
        float val = input[i * stride];
        largest = fmaxf(largest, val);
    }

    // Calculate exponentials and sum them up
    for(int i = 0; i < n; ++i){
        float e = expf((input[i * stride] - largest) / temp);
        sum += e;
        output[i * stride] = e;
    }

    // Normalize the results
    float inv_sum = 1.0f / sum; // Compute inverse sum once
    for(int i = 0; i < n; ++i){
        output[i * stride] *= inv_sum; // Use multiplication instead of division
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    // Calculate unique thread id
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x; 
    if (id >= batch * groups) return;

    int b = id / groups;
    int g = id % groups;

    // Pass the input and output pointers to softmax_device function
    softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}