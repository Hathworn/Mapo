#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int b = id / c;
    int k = id % c;
    int out_index = k + c * b;

    // Utilizing shared memory for better performance
    extern __shared__ float temp[];
    temp[threadIdx.x] = 0;

    for(int i = threadIdx.x; i < w*h; i += blockDim.x) {
        int in_index = i + h*w*(k + b*c);
        temp[threadIdx.x] += input[in_index];
    }
    __syncthreads();

    // Perform reduction within a block
    if (threadIdx.x == 0) {
        float sum = 0;
        for (int i = 0; i < blockDim.x; ++i) {
            sum += temp[i];
        }
        output[out_index] = sum / (w * h);
    }
}