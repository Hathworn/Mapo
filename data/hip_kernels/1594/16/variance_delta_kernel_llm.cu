#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    
    float var = variance[i] + .00001f;
    float var_factor = -.5f * powf(var, (float)(-3.f/2.f));
    
    float temp_variance_delta = 0;
    for(int j = 0; j < batch; ++j){
        for(int k = 0; k < spatial; ++k){
            int index = j * filters * spatial + i * spatial + k;
            temp_variance_delta += delta[index] * (x[index] - mean[i]);
        }
    }
    
    variance_delta[i] = temp_variance_delta * var_factor;
}