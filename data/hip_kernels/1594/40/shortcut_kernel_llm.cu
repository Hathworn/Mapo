#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    // Use a 2D block and grid to potentially improve memory coalescing and occupancy
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    int id = idy * (gridDim.x * blockDim.x) + idx; // compute global thread ID
    if (id >= size) return;

    int i = id % minw;
    id /= minw;
    int j = id % minh;
    id /= minh;
    int k = id % minc;
    id /= minc;
    int b = id % batch;

    int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
    int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));
    
    // Use registers for temporary storage to optimize memory accesses
    float out_val = s1 * out[out_index];
    float add_val = s2 * add[add_index];

    out[out_index] = out_val + add_val;
}