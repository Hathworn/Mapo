#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Use shared memory for variance to improve memory access performance
    extern __shared__ float shared_variance[];
    
    float scale = 1.f/(batch * spatial - 1);
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize shared variance to zero
    shared_variance[threadIdx.x] = 0;

    if (i < filters) {
        for (int j = 0; j < batch; ++j) {
            for (int k = 0; k < spatial; ++k) {
                int index = j * filters * spatial + i * spatial + k;
                shared_variance[threadIdx.x] += powf((x[index] - mean[i]), 2);
            }
        }
        // Move result from shared memory to global memory
        variance[i] = shared_variance[threadIdx.x] * scale;
    }
}