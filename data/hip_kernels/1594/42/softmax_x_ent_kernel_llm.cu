#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify grid calculation
    if(i < n){
        float p = pred[i];
        float t = truth[i];
        // Use of ternary operator for concise error calculation
        error[i] = t ? -__logf(p) : 0;  // Use __logf for faster computation
        delta[i] = t - p;
    }
}