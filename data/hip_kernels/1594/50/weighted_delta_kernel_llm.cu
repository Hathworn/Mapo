#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * gridDim.x) * blockDim.x; 

    // Check within bounds
    if(i < n)
    {
        float dc_val = dc[i]; // Cache dc[i] value
        float s_val = s[i];   // Cache s[i] value

        // Update da if da is not null
        if(da) 
            da[i] += dc_val * s_val;
        
        // Update db if db is not null
        if(db) 
            db[i] += dc_val * (1 - s_val);

        // Update ds
        ds[i] += dc_val * (a[i] - b[i]);
    }
}