#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    // Calculate the output dimensions
    int h = (in_h + pad - size)/stride + 1;
    int w = (in_w + pad - size)/stride + 1;
    int c = in_c;

    // Calculate the global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int b = id / (w * h * c);
    int k = (id / (w * h)) % c;
    int i = (id / w) % h;
    int j = id % w;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    // Calculate output index
    int out_index = j + w * (i + h * (k + c * b));
    
    // Initialize max values
    float max_val = -INFINITY;
    int max_idx = -1;

    // Iterate over pooling window
    for (int l = 0; l < size; ++l) {
        for (int m = 0; m < size; ++m) {
            int cur_h = h_offset + i * stride + l;
            int cur_w = w_offset + j * stride + m;
            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            
            // Check if the current position is valid
            bool valid = (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w);
            float current_val = valid ? input[index] : -INFINITY;
            
            // Update max values
            if (current_val > max_val) {
                max_idx = index;
                max_val = current_val;
            }
        }
    }

    // Store results
    output[out_index] = max_val;
    indexes[out_index] = max_idx;
}