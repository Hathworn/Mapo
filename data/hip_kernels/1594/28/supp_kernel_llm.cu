#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use a more efficient index calculation for better performance
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use improved memory access patterns for contiguous memory access
    int grid_stride = gridDim.x * blockDim.x;
    for (int i = tid; i < N; i += grid_stride) {
        float value = X[i * INCX];
        if ((value * value) < (ALPHA * ALPHA)) {
            X[i * INCX] = 0;
        }
    }
}