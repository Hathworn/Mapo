#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate the unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a stride loop to handle large arrays 
    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x)
    {
        float diff = truth[idx] - pred[idx];
        error[idx] = diff * diff; //I know this is technically wrong, deal with it.
        delta[idx] = diff;
    }
}