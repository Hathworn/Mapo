#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = gridDim.x * blockDim.x;
    
    // Unroll loop for better performance
    for (int idx = i; idx < (NX + NY) * B; idx += totalThreads) {
        int b = idx / (NX + NY);
        int j = idx % (NX + NY);
        if (j < NX) {
            OUT[idx] = X[b * NX + j];
        } else {
            OUT[idx] = Y[b * NY + j - NX];
        }
    }
}