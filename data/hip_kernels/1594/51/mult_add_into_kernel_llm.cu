#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we don't access out of bounds
    if (i < n)
    {
        // Perform the multiplication and addition
        c[i] += a[i] * b[i];
    }
}