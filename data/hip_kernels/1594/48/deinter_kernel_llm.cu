#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation
    int total = (NX + NY) * B;

    if(i < total){
        int b = i / (NX + NY);
        int j = i % (NX + NY);

        // Use ternary operator for cleaner if-else logic
        if (j < NX) {
            if (X)  X[b * NX + j] += OUT[i];
        } else {
            if (Y)  Y[b * NY + j - NX] += OUT[i];
        }
    }
}