#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int j;
    for(j = 0; j < batch; ++j){
        // Using unrolled loop for improved performance
        for(int i = id; i < spatial; i += threads){
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += x[index];
        }
    }

    __syncthreads();

    // Reduce sum using parallel reduction
    for(int stride = threads / 2; stride > 0; stride /= 2) {
        if(id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    if(id == 0){
        mean[filter] = local[0] / (spatial * batch);
    }
}