#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    // Calculate unique global thread index
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= N) return;
    
    // Efficiently calculate output indices
    int temp = i;
    int out_w = temp % (w * stride);
    temp /= (w * stride);
    int out_h = temp % (h * stride);
    temp /= (h * stride);
    
    // Compute batch and channel indices
    int out_c = temp % c;
    int b = temp / c;
    
    // Map output indices to input indices
    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_c = out_c;

    // Calculate flattened input index
    int in_index = b * w * h * c + in_c * w * h + in_h * w + in_w;

    // Perform the upsample operation with branching on forward mode
    if (forward)
        out[i] += scale * x[in_index];
    else
        atomicAdd(&x[in_index], scale * out[i]);
}