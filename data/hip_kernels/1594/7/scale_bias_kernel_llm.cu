#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    // Calculate the global index once outside the if condition
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Use globalIndex to eliminate separate calculation of offset
    if(globalIndex < size) 
    {
        int outputIndex = (batch * n + filter) * size + globalIndex; // Compute index
        output[outputIndex] *= biases[filter]; // Apply bias
    }
}