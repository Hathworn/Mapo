#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index directly, avoiding redundant grid calculations
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Step through the vector with strided increments if necessary
    for (int i = idx; i < N; i += blockDim.x * gridDim.x) {
        X[i * INCX] *= ALPHA;
    }
}