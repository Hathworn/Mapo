#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a stride loop to allow each thread to process multiple elements
    for (; i < n; i += blockDim.x * gridDim.x) {
        binary[i] = (x[i] >= 0) ? 1 : -1;
    }
}