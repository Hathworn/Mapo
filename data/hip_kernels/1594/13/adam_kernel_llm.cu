#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel_optimized(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (index >= N) return;

    // Pre-calculate common expressions outside memory access
    float one_minus_B1_pow_t = 1.f - powf(B1, t);
    float one_minus_B2_pow_t = 1.f - powf(B2, t);

    // Cache intermediate results to improve performance
    float m_i = m[index];
    float v_i = v[index];

    float mhat = m_i / one_minus_B1_pow_t;
    float vhat = v_i / one_minus_B2_pow_t;

    x[index] += rate * mhat / (sqrtf(vhat) + eps); // Combine assignment with calculation
}