#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D blockIdx
    if(id >= n) return;

    int k = id % c;
    int b = id / c; // Use integer division

    int out_index = k + c * b;
    float out_delta_val = out_delta[out_index] / (w * h); // Precompute constant division
    for(int i = 0; i < w * h; ++i){
        int in_index = i + h * w * out_index; // Reuse computed out_index
        atomicAdd(&in_delta[in_index], out_delta_val); // Use atomicAdd for concurrent writes
    }
}