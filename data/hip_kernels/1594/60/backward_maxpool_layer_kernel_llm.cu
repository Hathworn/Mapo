#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int c = in_c;
    int area = (size - 1) / stride;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    float d = 0;
    int out_w_start = (j - w_offset) / stride - area;
    int out_h_start = (i - h_offset) / stride - area;
    int out_w_end = out_w_start + 2 * area + 1;
    int out_h_end = out_h_start + 2 * area + 1;

    // Restrict the range to valid output indices
    out_w_start = max(out_w_start, 0);
    out_w_end = min(out_w_end, w);
    out_h_start = max(out_h_start, 0);
    out_h_end = min(out_h_end, h);

    // Compute deltas
    for (int out_h = out_h_start; out_h < out_h_end; ++out_h) {
        for (int out_w = out_w_start; out_w < out_w_end; ++out_w) {
            int out_index = out_w + w * (out_h + h * (k + c * b));
            if (indexes[out_index] == id) {
                d += delta[out_index];
            }
        }
    }
    prev_delta[id] += d;
}