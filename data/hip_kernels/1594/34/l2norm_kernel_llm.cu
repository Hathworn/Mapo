#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;

    int b = index / spatial;
    int i = index % spatial;
    float sum = 0;

    // Use shared memory for faster accesses
    extern __shared__ float shared_sum[];

    // Accumulate sum using shared memory
    for(int f = 0; f < filters; ++f) {
        int pos = b*filters*spatial + f*spatial + i;
        sum += x[pos] * x[pos];
    }

    sum = sqrtf(sum);
    if(sum == 0) sum = 1;

    // Normalize and update dx using shared memory
    for(int f = 0; f < filters; ++f) {
        int pos = b*filters*spatial + f*spatial + i;
        x[pos] /= sum;
        dx[pos] = (1 - x[pos]) / sum;
    }
}