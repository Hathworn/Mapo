#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation
    if(i < N) X[i*INCX] += ALPHA; // Simplified if condition
}