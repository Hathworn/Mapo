#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if(i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);
        if(abs_val < 1) {
            error[i] = diff * diff;
            delta[i] = diff;
        }
        else {
            error[i] = 2.0f * abs_val - 1.0f; // Use float literals for consistency
            delta[i] = copysignf(1.0f, diff); // Simplify sign assignment using copysignf
        }
    }
}