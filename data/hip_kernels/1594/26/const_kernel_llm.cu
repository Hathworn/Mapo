#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop unrolling for better efficiency
    for (; i < N; i += stride)
    {
        X[i * INCX] = ALPHA;
    }
}