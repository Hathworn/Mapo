#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;
    
    // Precompute common expressions to reduce calculations
    const int hw = w * h;
    const int chw = c * hw;
    
    // Determine the indexes within the multi-dimensional data
    int b = idx / chw;
    int channel_remainder = idx % chw;
    
    int in_c = channel_remainder / hw;
    int hw_remainder = channel_remainder % hw;
    
    int in_h = hw_remainder / w;
    int in_w = hw_remainder % w;
    
    // Compute output channel and corresponding offset
    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    
    // Calculate position in the output based on the stride
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    
    // Compute the output index using precomputed constants to improve indexing efficiency
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));
    
    // Forward or backward operation
    if (forward) 
        out[out_index] = x[idx];
    else 
        out[idx] = x[out_index];
}