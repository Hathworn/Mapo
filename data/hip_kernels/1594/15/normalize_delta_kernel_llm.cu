#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (index < N) {
        int f = (index / spatial) % filters;

        // Precompute repeated values for optimization
        float inv_sqrt_var = 1.f / sqrtf(variance[f] + .00001f);
        float factor = 2.f / (spatial * batch);
        
        // Update delta using precomputed values
        delta[index] = delta[index] * inv_sqrt_var + variance_delta[f] * factor * (x[index] - mean[f]) + mean_delta[f] / (spatial * batch);
    }
}