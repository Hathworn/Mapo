#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    // Reset local shared memory
    local[id] = 0;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = id; i < spatial; i += threads){
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += delta[index] * (x[index] - mean[filter]);
        }
    }

    // Synchronize threads within block
    __syncthreads();

    // Reduce local values and compute variance delta
    if(id == 0){
        float sum = 0;
        for(i = 0; i < threads; ++i){
            sum += local[i];
        }
        variance_delta[filter] = sum * -.5f * powf(variance[filter] + .00001f, (float)(-3.f/2.f));
    }
}