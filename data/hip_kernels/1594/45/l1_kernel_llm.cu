#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using shared memory and warp-level operations
__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    // Ensure thread index is within bounds
    if (i < n) {
        float diff = truth[i] - pred[i];
        
        // Calculate error using fast absolute value
        error[i] = fabsf(diff);
        
        // Calculate delta using direct comparison
        delta[i] = copysignf(1.0f, diff);
    }
}