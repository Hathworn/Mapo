#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified grid index calculation
    if (s >= size) return;

    float mean = 0.0f;

    // Use shared memory for mean calculation for better performance
    extern __shared__ float shared_mean[];
    if (threadIdx.x == 0) {
        for (int i = 0; i < n; ++i) {
            shared_mean[0] += fabsf(input[i * size + s]);
        }
        shared_mean[0] /= n;
    }
    __syncthreads();  // Ensure all threads have the computed mean

    mean = shared_mean[0];

    // Binarize inputs using computed mean
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0.0f) ? mean : -mean;
    }
}