#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_offset(int *key, int *idx, int *offset, int size) {

    int idxX = threadIdx.x + blockIdx.x * blockDim.x;

    // Use shared memory for storing temporary values to minimize global memory reads
    __shared__ int sharedKey[256]; // Assume blockDim.x <= 256

    // Load data into shared memory
    if (idxX < size) {
        sharedKey[threadIdx.x] = key[idxX];
    }
    __syncthreads();

    // Perform initialization only once if the block contains idxX == 0
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        offset[1] = 0;
    }

    if (idxX < size) {
        int keyVal = sharedKey[threadIdx.x];
        if (threadIdx.x == 0 && idxX > 0) {
            int keyValPrev = key[idxX - 1]; // Read previous key from global memory
            if (keyVal != keyValPrev) {
                offset[keyVal + 1] = idxX;
            }
        }
        else if (threadIdx.x > 0) {
            int keyValPrev = sharedKey[threadIdx.x - 1];
            if (keyVal != keyValPrev) {
                offset[keyVal + 1] = idxX;
            }
        }
    }

    // Perform final offset setup only for the last element
    if (idxX == size - 1) {
        int keyVal = sharedKey[threadIdx.x];
        offset[0] = keyVal + 1;
        offset[keyVal + 2] = size;
    }
}