#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Adds two arrays
extern "C" void runCudaPart();

// Main CUDA function
__global__ void addAry(int *ary1, int *ary2) {
    // Calculate global thread index
    int indx = blockIdx.x * blockDim.x + threadIdx.x;
    // Perform the addition
    ary1[indx] += ary2[indx];
}