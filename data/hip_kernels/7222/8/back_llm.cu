#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void back(double *h_out_d, double *weights_out_d, double *weights_h_d, double *weights_in_d, double *outputs_d, double *deltas_h_d, double *deltas_h_new_d, double *deltas_o_d, double *weights_in_delta_d, double *weights_out_delta_d, double *weights_h_delta_d, int height, int inputs, int outputs, int layers, double *training_in_d, double *training_out_d, int sample){

    int i, j;

    int tix = threadIdx.x;
    int tiy = threadIdx.y + sample;

    int h_offset = tiy * layers * height;
    int w_o_d_offset = tiy * outputs * height;
    int w_h_d_offset = tiy * (layers-1) * height * height;
    int w_i_d_offset = tiy * inputs * height;
    int d_h_offset = tiy * height;

    double delta_sum, temp;

    // Output layer processing
    if(tix < outputs){
        deltas_o_d[tiy * outputs + tix] = (outputs_d[tiy * outputs + tix] - training_out_d[tiy]);
        temp = deltas_o_d[tiy * outputs + tix];
        for(i = 0; i < height; i++){
            weights_out_delta_d[w_o_d_offset + (tix * height) + i] = temp * h_out_d[h_offset + (layers-1)*height+i];
        }
    }

    __syncthreads();

    // Hidden layer connected to output
    if(tix < height) {
        delta_sum = 0;
        for(i = 0; i < outputs; i++){
            delta_sum += weights_out_d[tix + (i * height)] * deltas_o_d[tiy * outputs + i];
        }
        temp = h_out_d[h_offset + (layers-1)*height + tix];
        deltas_h_d[d_h_offset + tix] = temp * (1 - temp) * delta_sum;

        for(i = 0; i < height; i++){
            weights_h_delta_d[w_h_d_offset + (layers-2)*height*height + (tix * height) + i] = deltas_h_d[d_h_offset + tix] * h_out_d[h_offset + (layers-2)*height+i];
        }
    }

    __syncthreads();

    // Each hidden layer not connected to input/hidden output layer
    for(i = layers - 2; i > 0; i--){
        if(tix < height) {
            delta_sum = 0;
            for(j = 0; j < height; j++){
                delta_sum += weights_h_d[i*height*height + j*height + tix] * deltas_h_d[d_h_offset + j];
            }
            temp = h_out_d[h_offset + i*height + tix];
            deltas_h_new_d[d_h_offset + tix] = temp * (1 - temp) * delta_sum;

            for(j = 0; j < height; j++){
                weights_h_delta_d[w_h_d_offset + (i-1)*height*height + (tix * height) + j] = deltas_h_new_d[d_h_offset + tix] * h_out_d[h_offset + (i-1)*height+j];
            }
        }

        __syncthreads();
        if(tix < height) {
            deltas_h_d[d_h_offset + tix] = deltas_h_new_d[d_h_offset + tix];
        }
        __syncthreads();
    }

    // Layer connected to inputs
    if(tix < height) {
        delta_sum = 0;
        for(i=0; i<height; i++){
            delta_sum += weights_h_d[i*height + tix] * deltas_h_d[d_h_offset + i];
        }
        temp = h_out_d[h_offset + tix];
        deltas_h_new_d[d_h_offset + tix] = temp * (1 - temp) * delta_sum;

        for(i=0; i<inputs; i++){
            weights_in_delta_d[w_i_d_offset + tix*inputs+i] = deltas_h_new_d[d_h_offset + tix] * training_in_d[tiy * inputs + i];
        }
    }

    __syncthreads();
}