#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update(double *weights_in_d, double *weights_h_d, double *weights_out_d, double *weights_in_delta_d, double *weights_h_delta_d, double *weights_out_delta_d, double *error_d) {
    int tix = threadIdx.x;
    int tid = blockIdx.x * blockDim.x + tix; // Calculate global thread index

    // Use shared memory to reduce global memory accesses
    __shared__ double shared_error;

    if (tix == 0) {
        shared_error = error_d[0];
    }

    __syncthreads(); // Synchronize to ensure shared_error is updated

    if (tid < INPUTS * H_HEIGHT) {
        weights_in_d[tid] -= (alpha_d * weights_in_delta_d[tid] / 55);
        weights_in_delta_d[tid] = 0.0;
    }

    if (tid < H_HEIGHT * (INPUTS + OUTPUTS)) {
        weights_h_d[tid] -= (alpha_d * weights_h_delta_d[tid] / 55);
        weights_h_delta_d[tid] = 0.0;
    }

    if (tid < OUTPUTS * H_HEIGHT) {
        weights_out_d[tid] -= (alpha_d * weights_out_delta_d[tid] / 55);
        weights_out_delta_d[tid] = 0.0;
    }

    if (tix == 0) {
        double local_error = shared_error * 100.0 / 55;
        printf("\nGPU Error: %f\n", local_error);
        error_d[0] = 0;
    }
}