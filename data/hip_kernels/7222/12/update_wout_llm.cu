#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_wout(double * weights_out_d, double *weights_out_delta_d, int bit){
    // Use shared memory for reduction
    __shared__ double weights_out_delta_s[OUTPUTS * H_HEIGHT];
    
    int tix = threadIdx.x;
    int tiy = threadIdx.y;
    int offset = OUTPUTS * H_HEIGHT;
    int index = tiy * offset + tix;

    // Load data into shared memory
    weights_out_delta_s[index] = weights_out_delta_d[index];
    __syncthreads();

    // Perform reduction in shared memory
    for(int s = blockDim.y / 2; s > 0; s >>= 1) {
        if(tiy < s) {
            weights_out_delta_s[index] += weights_out_delta_s[index + s * offset];
        }
        __syncthreads();
    }

    // Update weights and reset delta
    if(tiy == 0){
        atomicAdd(&weights_out_d[tix], -(alpha_d * weights_out_delta_s[tix] / (true_sample * 55.0)));
    }
    __syncthreads();
    if (index < offset) { // Ensure the index is valid for reset
        weights_out_delta_d[index] = 0.0;
    }
}