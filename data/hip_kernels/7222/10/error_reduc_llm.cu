#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void error_reduc(double *error_d, int bit) {
    __shared__ double error_ds[55];
    int tix = threadIdx.x;

    // Load shared memory in parallel
    if (tix < 55) {
        error_ds[tix] = error_d[tix];
    }
    __syncthreads();

    // Use parallel reduction with a stride
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tix < s) {
            error_ds[tix] += error_ds[tix + s];
        }
        __syncthreads();
    }

    // Normalize and output result from the first thread
    if (tix == 0) {
        error_ds[0] /= 55.0;
        printf("GPU Error: %f\n", error_ds[0] * 100.0);
    }

    // Reset the error array
    if (tix < 55) {
        error_d[tix] = 0.0;
    }
}