#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
//const int L = ;
const int scheme = 1; // 0 for explicit, 1 for implicit

/*************************************************************************************************************/
/*************								INITIALIZATION CODE										**********/
/*************************************************************************************************************/

__global__ void newForceReduction(float *input, float *output, int startunit, int len)
{
    unsigned int tx = threadIdx.x;
    unsigned int start = blockIdx.x * N_ATOMS;
    __shared__ float partSum[BLOCK_SIZE];

    // Load input elements into shared memory and handle out-of-bounds threads
    partSum[tx] = (tx < N_ATOMS) ? input[start + tx] : 0.0f;
    __syncthreads();

    // Optimized reduction pattern with loop unrolling
    for (unsigned int stride = BLOCK_SIZE / 2; stride > 32; stride >>= 1) {
        if (tx < stride) {
            partSum[tx] += partSum[tx + stride];
        }
        __syncthreads();
    }

    // Unrolled final warp for better performance without __syncthreads()
    if (tx < 32) {
        partSum[tx] += partSum[tx + 32];
        partSum[tx] += partSum[tx + 16];
        partSum[tx] += partSum[tx + 8];
        partSum[tx] += partSum[tx + 4];
        partSum[tx] += partSum[tx + 2];
        partSum[tx] += partSum[tx + 1];
    }

    // Write result to output array from the first thread in the block
    if (tx == 0) {
        output[blockIdx.x] = -partSum[0];
    }
}