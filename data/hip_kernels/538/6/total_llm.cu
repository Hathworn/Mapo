#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
const int scheme = 1; // 0 for explicit, 1 for implicit

__global__ void total(float *input, float *output, int len)
{
    // Shared memory for storing partial sums
    __shared__ float partSum[2 * BLOCK_SIZE];
    unsigned int tx = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * BLOCK_SIZE;

    // Load input floats to shared memory with boundary checks
    partSum[tx] = (start + tx < len) ? input[start + tx] : 0;
    partSum[BLOCK_SIZE + tx] = (start + BLOCK_SIZE + tx < len) ? input[start + BLOCK_SIZE + tx] : 0;

    // Reduce using a binary reduction tree approach
    for (unsigned int stride = BLOCK_SIZE; stride > 0; stride >>= 1) {
        __syncthreads();
        if (tx < stride) {
            partSum[tx] += partSum[tx + stride];
        }
    }

    // Output the block sum
    if (tx == 0) { 
        output[blockIdx.x] = partSum[0]; 
    }
}