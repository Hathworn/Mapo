#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
const int scheme = 1; // 0 for explicit, 1 for implicit

// Optimized kernel function for force reduction
__global__ void forcered_simple(float *force, float *forcered) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    __shared__ float forcered_sh[3 * N_ATOMS];

    // Initialize shared memory
    for (int j = threadIdx.x; j < 3 * N_ATOMS; j += blockDim.x) {
        forcered_sh[j] = 0.0f;
    }
    __syncthreads();

    // Perform force reduction using shared memory
    if (index < 3 * N_ATOMS) {
        int findex = (index / N_ATOMS) * N_ATOMS * N_ATOMS + index % N_ATOMS;
        for (int i = 0; i < N_ATOMS; i++) {
            forcered_sh[index] += force[findex + i * N_ATOMS];
        }
    }
    __syncthreads();

    // Write result back to global memory
    if (index < 3 * N_ATOMS) {
        forcered[index] = forcered_sh[index];
    }
}