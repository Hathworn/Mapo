#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
const int scheme = 1; // 0 for explicit, 1 for implicit

__device__ float PutInBox(float r) {
    if (fabs(r) > L / 2.0)
        r += (2 * (r < 0) - 1) * ceil((fabs(r) - L / 2.0f) / L) * L;
    return r;
}

__global__ void kinematics_phase1(float* positions, float* force, float* vel, int len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < len) {
        // Simplify calculation by reusing computed values
        float tempa = force[index] / MASS_ATOM;
        float tempvel = vel[index];
        float tempr = positions[index] + 0.5f * tempa * time_step * time_step + tempvel * time_step;
        
        // Ensure operations minimize redundant memory access
        positions[index] = PutInBox(tempr);
        vel[index] = tempvel + 0.5f * tempa * time_step;
    }
}