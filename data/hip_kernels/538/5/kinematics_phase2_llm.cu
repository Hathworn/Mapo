#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
const int scheme = 1; // 0 for explicit, 1 for implicit

/*************************************************************************************************************/
/*************								INITIALIZATION CODE										**********/
/*************************************************************************************************************/

__global__ void kinematics_phase2(float* force, float* vel, int len) {
  // Use thread-per-element pattern with bounds check
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < len) {
    vel[index] += 0.5f * force[index] / MASS_ATOM * time_step; // Simplified computation for better performance
  }
}