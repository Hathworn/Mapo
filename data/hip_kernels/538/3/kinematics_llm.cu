#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// Include files

// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
// const int L = ;
const int scheme = 1; // 0 for explicit, 1 for implicit

/*************************************************************************************************************/
/*************								INITIALIZATION CODE										**********/
/*************************************************************************************************************/

__device__ float PutInBox(float r) {
    if (fabs(r) > L / 2.0)
        r += (2 * (r < 0) - 1) * ceil((fabs(r) - L / 2.0f) / L) * L;
    return r;
}

__global__ void kinematics(float* positions, float* force, float* vel, int len) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within the limit
    if (index < len) {
        float mass_inv = 1.0f / MASS_ATOM;  // Pre-calculate inverse of mass
        float half_step_mass_inv = 0.5f * time_step * time_step * mass_inv;
        float vel_factor = time_step * mass_inv;

        // Calculate new position
        float new_pos = positions[index] + half_step_mass_inv * force[index] + vel[index] * time_step;
        positions[index] = PutInBox(new_pos);

        // Update velocity
        vel[index] += vel_factor * force[index];
    }
}