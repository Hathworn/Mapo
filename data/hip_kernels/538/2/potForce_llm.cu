#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
const int scheme = 1; // 0 for explicit, 1 for implicit

__global__ void potForce(float * PairWise, int N, float * PotOut, float * ForceOut)
{
    int row = threadIdx.x + blockIdx.x * blockDim.x;

    // Return early if out of bounds
    if (row >= N * N) return;

    // Load data into registers to minimize global memory access
    float delx = PairWise[row];
    float dely = PairWise[row + N * N];
    float delz = PairWise[row + 2 * N * N];

    float delr2 = delx * delx + dely * dely + delz * delz;
    // Avoid division/zero errors
    if (delr2 == 0.0f) {
        PotOut[row] = 0.0f;
        ForceOut[row] = 0.0f;
        ForceOut[row + N * N] = 0.0f;
        ForceOut[row + 2 * N * N] = 0.0f;
        return;
    }

    float delrm6 = rsqrtf(delr2 * delr2 * delr2);
    float delrm12 = delrm6 * delrm6;
    float potential_val = 4 * (delrm12 - delrm6);
    PotOut[row] = potential_val;

    // Pre-compute common factor for forces
    float force_factor = -24 * (2 * delrm12 - delrm6) / delr2;
    
    ForceOut[row] = delx * force_factor;
    ForceOut[row + N * N] = dely * force_factor;
    ForceOut[row + 2 * N * N] = delz * force_factor;
}