#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel with thread check
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    if (threadIdx.x == 0) { // Ensure only one thread performs the division
        *accuracy /= N;
    }
}