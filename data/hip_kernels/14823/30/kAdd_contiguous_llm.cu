#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Use a single loop iteration per thread for better readability and performance
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEls) {
        dest[idx] = a[idx] + b[idx];
    }
}