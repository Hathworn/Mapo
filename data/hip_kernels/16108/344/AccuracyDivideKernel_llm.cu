#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// Optimized kernel to prevent race conditions using atomic operations
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    if (threadIdx.x == 0) {  // Single thread performs division
        atomicExch(accuracy, *accuracy / N);
    }
}