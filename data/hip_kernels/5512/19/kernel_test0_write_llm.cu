#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);

    // Ensure early exit for out-of-bound blocks
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Use loop unrolling for performance improvement
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        ptr[i] = pattern;
    }
}