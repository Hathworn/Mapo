#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    // Calculate starting pointer with bounds check
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);
    if (ptr >= (unsigned int*)end_ptr) {
        return; // Exit early if out of bounds
    }

    // Use threadIdx.x to parallelize writing pattern across multiple threads
    int tid = threadIdx.x;
    for (unsigned int i = tid; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        ptr[i] = pattern; // Write pattern to each calculated position
    }

    return;
}