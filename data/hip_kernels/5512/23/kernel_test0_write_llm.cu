#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    unsigned int* orig_ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* ptr = orig_ptr;
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);
    unsigned int pattern = 1;
    unsigned long mask = 4;

    *ptr = pattern;

    while(ptr < block_end){

        // This operation updates ptr using bitwise OR with mask
        ptr = (unsigned int*) (((unsigned long)orig_ptr) | mask);

        // When ptr equals orig_ptr, increment mask and continue to avoid infinite loop
        if (ptr == orig_ptr){
            mask = mask << 1;
            continue;
        }
        // Check if ptr exceeds the block_end to exit the loop
        if (ptr >= block_end){
            break;
        }

        *ptr = pattern;

        // Update pattern and mask for next iteration
        pattern = pattern << 1;
        mask = mask << 1;
    }
    return;
}