#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    // Calculate thread index and make pointer arithmetic safer
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int* ptr = (unsigned int*) (_ptr + tid * BLOCKSIZE);
    unsigned int* end_ptr_uint = (unsigned int*) end_ptr;

    if (ptr >= end_ptr_uint) {
        return;
    }

    unsigned int half_count = BLOCKSIZE / sizeof(unsigned int) / 2;
    unsigned int* ptr_mid = ptr + half_count;

    // Optimize memory copy using loop unrolling
    #pragma unroll
    for (unsigned int i = 0; i < half_count; i++) {
        ptr_mid[i] = ptr[i];
    }

    #pragma unroll
    for (unsigned int i = 0; i < half_count - 8; i++) {
        ptr[i + 8] = ptr_mid[i];
    }

    #pragma unroll
    for (unsigned int i = 0; i < 8; i++) {
        ptr[i] = ptr_mid[half_count - 8 + i];
    }
    
    return;
}