#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    // Calculate and cache the starting pointer for the current block
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check if the starting pointer exceeds the end_ptr
    if (ptr >= (unsigned long*) end_ptr) {
        return;
    }

    // Use threadIdx.x to access each element in parallel
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE/sizeof(unsigned long); i += blockDim.x) {
        ptr[i] = (unsigned long) &ptr[i];
    }
}