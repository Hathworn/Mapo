#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check if the block's data region is valid
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Calculate the number of iterations before the loop
    unsigned int limit = BLOCKSIZE / sizeof(unsigned int);
    
    // Use fewer iterations for setting p1 and p2 to reduce overhead
    unsigned int i;
    for (i = offset; i < limit; i += MOD_SZ){
        ptr[i] = p1;
    }

    for (i = 0; i < limit; i++){
        if (i % MOD_SZ != offset){
            ptr[i] = p2;
        }
    }

    return;
}