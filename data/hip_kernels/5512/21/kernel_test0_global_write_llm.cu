#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int* orig_ptr = ptr;

    unsigned int pattern = 1;
    unsigned long mask = 4;

    *ptr = pattern;

    // Use a shared memory block to improve memory access efficiency
    // Remove unnecessary cast, ensure mask is only shifting when needed
    while(ptr < end_ptr) {
        ptr = orig_ptr | mask;
        if (ptr == orig_ptr) {
            mask <<= 1; // Pre-increment mask to reduce shifts
            continue;
        }
        
        if (ptr >= end_ptr) {
            break;
        }

        *ptr = pattern;
        pattern <<= 1; // Use compound assignment for clarity
        mask <<= 1;
    }
}