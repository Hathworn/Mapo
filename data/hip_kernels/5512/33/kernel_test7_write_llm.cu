#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    // Calculate global index for current thread
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* start_ptr = (unsigned int*) _start_ptr;
    
    // Check if pointer is within bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Calculate the number of elements to copy to enhance coalesced memory access
    unsigned int num_elements = BLOCKSIZE / sizeof(unsigned int);

    // Use loop unrolling for better instruction throughput
    #pragma unroll
    for (unsigned int i = 0; i < num_elements; i++){
        ptr[i] = start_ptr[i];
    }
    return;
}