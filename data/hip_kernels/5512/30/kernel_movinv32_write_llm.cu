#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Early return if the start pointer is outside the valid range
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int index = threadIdx.x;
    unsigned int stride = blockDim.x;
    unsigned int k = offset + index;  // Adjust start position based on thread index
    unsigned int pat = pattern;

    // Unroll loop across threads in block to utilize warp and coalesced memory access
    for (unsigned int i = index; i < BLOCKSIZE / sizeof(unsigned int); i += stride) {
        ptr[i] = pat;
        k++;
        if (k >= 32) {
            k = 0;
            pat = lb;
        } else {
            pat = pat << 1;
            pat |= sval;
        }
    }
}