#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_init(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Early exit if ptr is out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int p1 = 1;
    #pragma unroll  // Enable loop unrolling
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i += 16) {
        unsigned int p2 = ~p1;

        // Use loop unrolling variable to centralize repeated code
        ptr[i] = p1; 
        ptr[i + 1] = p1; 
        ptr[i + 2] = p2; 
        ptr[i + 3] = p2; 
        ptr[i + 4] = p1; 
        ptr[i + 5] = p1; 
        ptr[i + 6] = p2; 
        ptr[i + 7] = p2; 
        ptr[i + 8] = p1; 
        ptr[i + 9] = p1; 
        ptr[i + 10] = p2; 
        ptr[i + 11] = p2; 
        ptr[i + 12] = p1; 
        ptr[i + 13] = p1; 
        ptr[i + 14] = p2; 
        ptr[i + 15] = p2;

        // Efficient modulo operation
        p1 = (p1 << 1) | (p1 >> 31);
    }
}