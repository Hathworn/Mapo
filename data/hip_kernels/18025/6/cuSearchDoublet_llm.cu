#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuSearchDoublet(const int* nSpM, const float* spMmat, const int* nSpB, const float* spBmat, const int* nSpT, const float* spTmat, const float* deltaRMin, const float* deltaRMax, const float* cotThetaMax, const float* collisionRegionMin, const float* collisionRegionMax, int*  nSpMcomp, int*  nSpBcompPerSpM_Max, int*  nSpTcompPerSpM_Max, int*  nSpBcompPerSpM, int*  nSpTcompPerSpM, int*  McompIndex, int*  BcompIndex, int*  tmpBcompIndex, int*  TcompIndex, int*  tmpTcompIndex ){

extern __shared__ float sharedMem[];
int* mPos      = (int*)sharedMem;
int* isMcompat = (int*)&mPos[1];

if (threadIdx.x==0) {
*isMcompat = false;
}
__syncthreads();

float rM = spMmat[blockIdx.x +(*nSpM)*3];
float zM = spMmat[blockIdx.x +(*nSpM)*2];

bool isBcompat, isTcompat;
int offset = threadIdx.x; // Initialize offset with threadIdx.x to avoid using separate offset varaible later.

while (offset < max(*nSpB, *nSpT)){

isBcompat = false;
isTcompat = false;

// Doublet search for bottom hits
if (offset < *nSpB ){
    float rB = spBmat[offset+(*nSpB)*3];
    float zB = spBmat[offset+(*nSpB)*2];
    float deltaR = rM - rB;
    float cotTheta = (zM - zB)/deltaR;

    if(deltaR <= *deltaRMax && deltaR >= *deltaRMin && fabsf(cotTheta) <= *cotThetaMax){
        float zOrigin = zM - rM*cotTheta;
        if (zOrigin >= *collisionRegionMin && zOrigin <= *collisionRegionMax) {
            isBcompat = true;
        }
    }

    if (isBcompat) {
        int bPos = atomicAdd(&nSpBcompPerSpM[blockIdx.x], 1);
        tmpBcompIndex[bPos+(*nSpB)*blockIdx.x] = offset;
    }
}

// Doublet search for top hits
if (offset < *nSpT){
    float rT = spTmat[offset+(*nSpT)*3];
    float zT = spTmat[offset+(*nSpT)*2];
    float deltaR = rT - rM;
    float cotTheta = (zT - zM)/deltaR;

    if (deltaR >= *deltaRMin && deltaR <= *deltaRMax && fabsf(cotTheta) <= *cotThetaMax){
        float zOrigin = zM - rM*cotTheta;
        if (zOrigin >= *collisionRegionMin && zOrigin <= *collisionRegionMax) {
            isTcompat = true;
        }
    }

    if (isTcompat){
        int tPos = atomicAdd(&nSpTcompPerSpM[blockIdx.x], 1);
        tmpTcompIndex[tPos+(*nSpT)*blockIdx.x] = offset;
    }
}

offset += blockDim.x; // Increment offset by blockDim.x
}

__syncthreads();

if (threadIdx.x == 0){
if (nSpBcompPerSpM[blockIdx.x] > 0 && nSpTcompPerSpM[blockIdx.x] > 0 ){
*mPos = atomicAdd(nSpMcomp,1);
*isMcompat = true;
McompIndex[*mPos] = blockIdx.x;

atomicMax(nSpBcompPerSpM_Max, nSpBcompPerSpM[blockIdx.x]);
atomicMax(nSpTcompPerSpM_Max, nSpTcompPerSpM[blockIdx.x]);
}
}

__syncthreads();

if (*isMcompat){
offset = threadIdx.x;

while(offset < max(nSpBcompPerSpM[blockIdx.x], nSpTcompPerSpM[blockIdx.x] ) ){

if (offset < nSpBcompPerSpM[blockIdx.x]){
BcompIndex[offset+(*nSpB)*(*mPos)]
= tmpBcompIndex[offset+(*nSpB)*blockIdx.x];
}

if (offset < nSpTcompPerSpM[blockIdx.x]){
TcompIndex[offset+(*nSpT)*(*mPos)]
= tmpTcompIndex[offset+(*nSpT)*blockIdx.x];
}
offset += blockDim.x;
}
}

}