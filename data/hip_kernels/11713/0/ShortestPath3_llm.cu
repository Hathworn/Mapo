#include "hip/hip_runtime.h"

```cpp
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCKSIZE 4
#define CELLS_PER_THREAD 4     // Stride length
__global__ void ShortestPath3(float *Arr1, float *Arr2, int N) {
    // Arr1 input array, holds weights
    // Arr2 output array
    unsigned int k;
    int row = blockIdx.x;
    int col = threadIdx.x;

    if (row >= N || col >= N) return;

    int index = row * N + col;                     // Index for elements of global memory array
    extern __shared__ float sArr[];                // Dynamic shared memory allocation in array
    int stride = threadIdx.x;                      // Adjust stride to threadIdx.x

    for (int i = 0; i < N; i++) {                  // Copy elements of the same row to shared memory
        if (i + stride * CELLS_PER_THREAD >= N) break; 
        sArr[i + stride * CELLS_PER_THREAD] = Arr1[index + i * N / CELLS_PER_THREAD];
    }
    __syncthreads();                                // Wait for all threads in the current block

    for (int i = 0; i < N; i++) {                  // Iterate through shared elements
        if (i == threadIdx.x + stride * CELLS_PER_THREAD) continue;
        if (sArr[threadIdx.x + stride * CELLS_PER_THREAD] > (sArr[i] + Arr1[i * N + col + stride * CELLS_PER_THREAD])) {
            Arr2[index + stride * CELLS_PER_THREAD] = (sArr[i] + Arr1[i * N + col + stride * CELLS_PER_THREAD]);
        }
    }
}