#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ShortestPath2(float *Arr1, float *Arr2, float *recv, int N, int rows, int k, int rank, int owner) {
    // Compute row and column for the current thread
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < N && row < rows) {  // Bounds checking to prevent out-of-bounds access
        int index = row * N + col;
        int index_ik = row * N + k;

        // Conditional update of Arr2 based on path comparison
        float potential_new_distance = Arr1[index_ik] + recv[col];
        if (Arr1[index] > potential_new_distance) {
            Arr2[index] = potential_new_distance;
        }
    }
}