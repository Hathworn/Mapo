#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCKSIZE 4
#define CELLS_PER_THREAD 4     // Stride length
__global__ void ShortestPath2(float *Arr1, float *Arr2, int N) {
    unsigned int k;

    int row = blockIdx.x;
    int col = threadIdx.x;
    if (row >= N || col >= N) return;

    int index = row * N + col;                  // Index for Elements of global memory array
    extern __shared__ float sArr[];             // Dynamic share memory allocation in Array
    sArr[col] = Arr1[index];                    // Copy elements of same ROW in shared memory (ThreadIdx.x as index)
    __syncthreads();                            // Wait threads in block to finish

    float current = sArr[col];
    for (k = 0; k < N; k++) {
        if (k == col) continue;                 // Skip self
        float newPath = sArr[k] + Arr1[k * N + col];
        if (current > newPath) {                // Update if new path is shorter
            current = newPath;
        }
    }
    Arr2[index] = current;                      // Write the shortest path to output
}