#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ShortestPath1(float *Arr1, float *Arr2, int N, int rows, int rank) {
    // Calculate column and row indices
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Compute global offset
    int offset = rows * rank;
    
    // Calculate the index for the current thread
    int index = row * N + col;
    
    // Initialize Arr2 with Arr1 values
    Arr2[index] = Arr1[index];
    
    // Iterate over rows assigned to this process
    for (int k = rank * rows; k < (rank + 1) * rows; ++k) {
        int index_ik = row * N + k;
        int index_kj = (k - offset) * N + col; // Correct offset usage
        
        // Perform path comparison and update
        if (Arr1[index] > (Arr1[index_ik] + Arr1[index_kj])) {
            Arr2[index] = Arr1[index_ik] + Arr1[index_kj];
        }
    }
    // Synchronize threads in the same block after each iteration
    __syncthreads(); // Synchronize threads in block
}