#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCKSIZE 4
#define CELLS_PER_THREAD 4     // Stride length

__global__ void ShortestPath1(float *Arr1, float *Arr2, int N) {
    // Arr1 input array, Holds of (u, v)
    // Arr2 output array
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = row * N + col;

    if ((row < N) && (col < N)) {
        Arr2[index] = Arr1[index];
        
        // Use shared memory to optimize global memory access
        __shared__ float sharedArr1[BLOCKSIZE][BLOCKSIZE];
        
        for (int k = 0; k < N; ++k) {
            sharedArr1[threadIdx.y][threadIdx.x] = Arr1[row * N + k];
            __syncthreads();
            
            for (int i = 0; i < BLOCKSIZE; ++i) {
                if (Arr2[index] > sharedArr1[threadIdx.y][i] + Arr1[i * N + col]) {
                    Arr2[index] = sharedArr1[threadIdx.y][i] + Arr1[i * N + col];
                }
            }
            __syncthreads();
        }
    }
}