#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*******************************************************copy the components to the wave**************************************************************/
/****************************************************normalize operation*************************************************************/
/****************************************************compute_masks operation*************************************************************/
__global__ void normalize(float *nor_ary, float *flit_ary, float tw, float ts, size_t N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory to reduce global memory access if N is large
    __shared__ float temp_ary[256]; // Example: assuming blockDim.x <= 256

    if (tid < N) {
        // Load data into shared memory
        temp_ary[threadIdx.x] = flit_ary[tid];
        __syncthreads();

        // Compute normalization using shared memory
        float value = temp_ary[threadIdx.x];
        if (value >= ts) nor_ary[tid] = 1.0f;
        else if (value < tw) nor_ary[tid] = 0.0f;
        else nor_ary[tid] = (value - tw) / (ts - tw);
    }
}