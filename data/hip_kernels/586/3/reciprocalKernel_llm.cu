#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
============================================================================
Name        : SpikeSorting.cu
Author      : John
Version     :
Copyright   :
Description : CUDA compute reciprocals
============================================================================
*/

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
    // Calculate a unique thread index
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform bounds checking before computation
    if (idx < vectorSize) {
        float value = data[idx]; // Avoid repeated global memory access
        data[idx] = 1.0 / value; // Compute reciprocal
    }
}