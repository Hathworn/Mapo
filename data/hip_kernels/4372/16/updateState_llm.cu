#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#include <hiprand_kernel.h> // Include HIPRAND library

__global__ void updateState(double *B, double *external, double *lamBeta, int dim, float timestep, double noise, int length, int totalIterations, int iterationNum) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x + length;
    if (index >= length && index < length + dim) {
        int neuronNum = index % dim;
        double lam = lamBeta[neuronNum * 2];
        double beta = lamBeta[neuronNum * 2 + 1];

        double input = B[index] + external[neuronNum * totalIterations + iterationNum];
        double old_output = B[index - dim];
        double d_layers = (-old_output) + 1.0 / (1.0 + exp(-lam * (input - beta))); // Simplified calculations

        // Use HIPRAND library for better random number generation
        hiprandState_t state;
        hiprand_init(blockIdx.x * blockDim.x + threadIdx.x, 0, 0, &state); // Simplified initialization
        double random = hiprand_normal_double(&state); // Use double precision 
        double gauss_noise = noise * random * sqrt(timestep);
        
        B[index] = old_output + d_layers * timestep + gauss_noise; // Update state
    }
}
```
