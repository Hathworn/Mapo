#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void squeeze(float *B, int dim, int length, float L, float M) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within the bounds
    if (index < dim) {
        index += length; // Offset index by length
        float value = B[index];
        // Apply transformation
        B[index] = 1.0f / (1.0f + __expf(-L * (value - M)));
    }
}