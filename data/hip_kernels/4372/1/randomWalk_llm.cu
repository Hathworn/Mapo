#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomWalk(double *results, int *crossTimes, int T, int N, double drift, int numSims, double lowerThreshold, double upperThreshold, int deviceID) {

    int crossTimeIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (crossTimeIndex < numSims) {
        // Initialize random state per thread using an improved seed for better randomness
        hiprandState_t state;
        hiprand_init(blockIdx.x * (1000ULL * deviceID) + threadIdx.x + clock64(), 0, 0, &state);
        
        double random;
        int start = (threadIdx.x + blockIdx.x * blockDim.x) * N;

        crossTimes[crossTimeIndex] = 0;
        results[start] = 0.0;
        bool crossed = false;
        double sqrt_dt = sqrt((double)T / N); // Precompute sqrt(T/N) for efficiency

        for (int j = start + 1; j < start + N; j++) {
            // Use single random call and precomputed sqrt_dt to enhance performance
            random = hiprand_normal_double(&state);
            results[j] = results[j-1] + drift * ((double)T / N) + random * sqrt_dt;
            
            if (!crossed && results[j] >= upperThreshold) {
                crossTimes[crossTimeIndex] = j - start;
                crossed = true;
            } else if (!crossed && results[j] <= lowerThreshold) {
                crossTimes[crossTimeIndex] = -1 * (j - start);
                crossed = true;
            }
        }
    }
}