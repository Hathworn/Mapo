#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel launch configuration using stride loop
__global__ void squeeze(float *B, int dim, int length, float L, float M) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x + length;
    int stride = gridDim.x * blockDim.x; // Calculate stride for loop unrolling
    while (index < length + dim) { // Use loop to calculate multiple elements per thread
        B[index] = 1 / (1 + expf(-L * (B[index] - M)));
        index += stride; // Increment index by stride
    }
}