#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Array to store crossing time of each simulation, positive time for upper threshold and negative time for lower threshold. 0 if never crossed.
int *crossTimes = nullptr;

// Data structure to hold information for each GPU
struct DataStruct {
    int deviceID;          // id of gpu
    int sims;              // number of simulations to be executed on this gpu
    double *resultArray;   // array to store brownian paths calculated on this gpu
    int *crossArray;       // array to store cross times calculates on this gpu
    int N;                 // number of simulations on this gpu
    double T;              // parameter for brownian path equation
    double drift;          // drift parameter brownian path equation
    double lowerThreshold;
    double upperThreshold;
};

// Optimized function to execute on each individual GPU
__global__ void randomWalk(double *results, int *crossTimes, int T, int N, double drift, int numSims, double lowerThreshold, double upperThreshold, double deviceID) {
    
    // Unique simulation index
    int simIndex = threadIdx.x + blockIdx.x * blockDim.x;

    // Check boundary condition early to avoid unnecessary computations
    if (simIndex >= numSims) return;

    // Initialize random number generator
    hiprandState_t state;
    hiprand_init(blockIdx.x * (1000 * deviceID) + threadIdx.x + clock64(), 0, 0, &state);

    // Starting position of this simulation in results array
    int start = simIndex * N;
    
    // Set default value of cross time for this simulation to 0
    crossTimes[simIndex] = 0;
    results[start] = 0.0;

    bool crossed = false;
    double sqrtT_N = sqrt((double) T / N); // Precompute sqrt to improve performance
    double drift_T_N = drift * ((double) T / N); // Precompute drift factor

    for (int j = 1; j < N; j++) { // Directly iterate over the steps

        // Generate next step of path
        double random = hiprand_normal_double(&state);
        results[start + j] = results[start + j - 1] + drift_T_N + random * sqrtT_N;

        // Store crossing time
        if (!crossed) {
            if (results[start + j] >= upperThreshold) {
                crossTimes[simIndex] = j;
                crossed = true;
            } else if (results[start + j] <= lowerThreshold) {
                crossTimes[simIndex] = -j;
                crossed = true;
            }
        }
    }
}