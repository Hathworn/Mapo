#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void changeValues(float *matrix, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Use local variable to store and truncate value before writing to global memory
    if (index < size) {
        float value = matrix[index];
        matrix[index] = ((int)(value * 10));
    }
}