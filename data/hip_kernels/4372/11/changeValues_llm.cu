#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void changeValues(float *matrix, int size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < size) {
        // Combine calculations to reduce temporary variables
        matrix[index] = (float)((int)(matrix[index] * 10));
    }
}