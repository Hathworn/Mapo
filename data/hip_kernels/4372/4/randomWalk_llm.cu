#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomWalk(double *results, int *crossTimes, int T, int N, int numSims) {
// Use 2D grid for improved index mapping
int crossTimeIndex = blockIdx.x * blockDim.x + threadIdx.x;

if (crossTimeIndex < numSims) {
    hiprandState_t state;
    // Initialize the random generator per thread for unique sequences
    hiprand_init(1234, crossTimeIndex, 0, &state);
    
    double random;
    int start = crossTimeIndex * N;  // Simplified access

    crossTimes[crossTimeIndex] = 0;
    results[start] = 0.0;

    // Use vectorized access for SIMD potential 
    for (int j = 1; j < N; j++) {   
        random = hiprand_normal_double(&state);
        results[start + j] = results[start + j - 1] + random * sqrt((double)T / N);
    }
}
}
```
