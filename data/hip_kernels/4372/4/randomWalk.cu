#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomWalk(double *results, int *crossTimes, int T, int N, int numSims) {
int crossTimeIndex = threadIdx.x + blockIdx.x * blockDim.x;
if (crossTimeIndex < numSims) {
hiprandState_t state;
hiprand_init (1234, 0, 0, &state);
double random;
int start = (threadIdx.x + blockIdx.x * blockDim.x) * N;

crossTimes[crossTimeIndex] = 0;
results[start] = 0.0;

for (int j = start + 1; j < start + N; j++) {
random = hiprand_normal_double(&state);
results[j] = results[j-1] + random * sqrt((double) T / N);
}
}

/*
Generate 2 doubles at once. Test later to see if this is more efficient:
double hiprand_normal2_double (state);
*/

}