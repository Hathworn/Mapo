#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getCrossingTimes(double *results, int *crossTimes, int N, int numSims, int lowerThreshold, int upperThreshold) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Pre-computed stride for all threads

    // Unrolling the loop for better performance
    while (tid < N * numSims) {
        int simIndex = tid / N;
        if (crossTimes[simIndex] == 0) {
            double result = results[tid];
            if (result <= lowerThreshold || result >= upperThreshold) {
                crossTimes[simIndex] = tid % N;
            }
        }
        tid += stride; // Use pre-computed stride instead of re-computing
    }
}