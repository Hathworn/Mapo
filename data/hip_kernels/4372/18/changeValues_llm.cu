#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void changeValues(float *matrix, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index is within bounds and process multiple elements per thread
    for (int i = index; i < size; i += blockDim.x * gridDim.x) {
        float a = matrix[i] * 10.0f;
        matrix[i] = floorf(a); // Direct conversion to integer using built-in function
    }
}