#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomWalk(double *results, int *crossTimes, int T, int N, int numSims, double upperThreshold, double deviceID) {

    // Calculate global thread index
    int crossTimeIndex = threadIdx.x + blockIdx.x * blockDim.x; 

    if (crossTimeIndex < numSims) {
        // Initialize random number generator with unique seed
        hiprandState_t state;
        hiprand_init(blockIdx.x * (1000 * deviceID) + threadIdx.x + clock64(), 0, 0, &state);

        double random;
        int start = crossTimeIndex * N; // Optimize computation for start index
        crossTimes[crossTimeIndex] = 0; 
        results[start] = 0.0; 
        bool crossed = false;

        // Unroll loop to enhance performance
        #pragma unroll 4 
        for (int j = 1; j < N; j++) {
            random = hiprand_normal_double(&state);
            results[start + j] = results[start + j - 1] + random * sqrt((double) T / N);

            // Check for crossing event
            if (!crossed && results[start + j] >= upperThreshold) {
                crossTimes[crossTimeIndex] = j;
                crossed = true;
            } 
        }
    }
}