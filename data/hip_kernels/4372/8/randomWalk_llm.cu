#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomWalk(double *results, int T, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x; // Calculate unique thread index
    if (idx >= N) return; // Ensure index does not exceed array bounds

    hiprandState_t state;
    hiprand_init(1234, idx, 0, &state); // Use unique index for random seed

    double random;

    if (idx == 0) {
        results[idx] = 0.0; // Initialize first element
    } else {
        random = hiprand_normal_double(&state);
        results[idx] = results[idx - 1] + random * sqrt((double)T / N); // Compute result
    }
}