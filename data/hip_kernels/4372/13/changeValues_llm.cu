#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void changeValues(float *matrix, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        // Combine operations and remove unnecessary casting
        matrix[index] = (int)(matrix[index] * 10);
    }
}