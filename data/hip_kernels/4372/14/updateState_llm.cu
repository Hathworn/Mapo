#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateState(float *B, float *external, int dim, float timestep, float noise, int length, int totalIterations, int iterationNum, float L, float M) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x + length;
    // Ensure index is within bounds
    if (index >= length && index < length + dim) {
        int neuronNum = index % dim;
        // Pre-compute reused values for efficiency
        float input = B[index] + external[neuronNum * totalIterations + iterationNum];
        float old_output = B[index - dim];
        float exp_val = expf(-1 * L * (input - M));
        float d_layers = (-1 * old_output) + 1 / (1 + exp_val);

        // Initialize random number generator once per thread to reduce overhead
        __shared__ hiprandState_t sharedState[256];
        if (threadIdx.x < 256) {
            hiprand_init(blockIdx.x * 1000 + threadIdx.x, 0, 0, &sharedState[threadIdx.x]);
        }
        __syncthreads();
        hiprandState_t localState = sharedState[threadIdx.x];
        float random = hiprand_normal(&localState);
        B[index] = old_output + d_layers * timestep + noise * random * sqrt(timestep);
    }
}