#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomWalk(double *results, int *crossTimes, int T, int N, int numSims, double lowerThreshold, double upperThreshold, int *dev_failCross, double seconds) {
    int crossTimeIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (crossTimeIndex < numSims) {
        hiprandState_t state;
        // Initialize random state with more unique seeds
        hiprand_init(blockIdx.x * blockDim.x + threadIdx.x + seconds, 0, 0, &state);
        double random;
        int start = crossTimeIndex * N; // Use crossTimeIndex for a clear offset calculation

        crossTimes[crossTimeIndex] = 0;
        results[start] = 0.0;
        bool crossed = false;

        // Pre-calculate constants outside the loop
        double sqrtVal = sqrt((double)T / N);

        for (int j = 1; j < N; j++) { // Optimize loop index
            random = hiprand_normal_double(&state);
            results[start + j] = results[start + j - 1] + random * sqrtVal; // Access results using start + j
            if (!crossed) {
                if (results[start + j] >= upperThreshold) {
                    crossTimes[crossTimeIndex] = j;
                    crossed = true;
                } else if (results[start + j] <= lowerThreshold) {
                    crossTimes[crossTimeIndex] = -j;
                    crossed = true;
                }
            }
        }

        if (!crossed) {
            atomicAdd(dev_failCross, 1);
        }
    }
}