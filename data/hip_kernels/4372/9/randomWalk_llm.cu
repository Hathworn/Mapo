#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#include "hiprand_kernel.h"

__global__ void randomWalk(double *results, int *crossTimes, int T, int N, int numSims, double upperThreshold, double deviceID) {

    // Calculate the unique index for each simulation
    int crossTimeIndex = threadIdx.x + blockIdx.x * blockDim.x;

    // Check if the current thread is within the number of simulations
    if (crossTimeIndex < numSims) {
        
        // Initialize random number generator state with a unique seed
        hiprandState_t state;
        hiprand_init(blockIdx.x * (1000 * deviceID) + threadIdx.x + clock64(), 0, 0, &state);
        
        // Initialize and calculate the starting position in the results array for this simulation
        int start = crossTimeIndex * N;
        crossTimes[crossTimeIndex] = 0; // Default crossing time is 0 - simulation hasn't crossed the threshold

        bool crossed = false; // Track if the threshold has been crossed
        results[start] = 0.0; // Starting point of path

        // Loop through steps to simulate the random walk
        for (int j = 1; j < N; j++) {
            double random = hiprand_normal_double(&state); // Generate a random number
            results[start + j] = results[start + j - 1] + random * sqrt((double) T / N); // Calculate next step

            // Record crossing time once the upper threshold is crossed
            if (!crossed && results[start + j] >= upperThreshold) {
                crossTimes[crossTimeIndex] = j;
                crossed = true;
            }
        }
    }
}