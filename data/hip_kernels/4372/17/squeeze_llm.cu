#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void squeeze(float *B, int dim, int length, float L, float M) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Compute actual global index
    int actualIndex = index + length;
    if (actualIndex < length + dim) {
        // Avoid repeated computation of -1 * L
        float tmp = -L * (B[actualIndex] - M);
        B[actualIndex] = 1 / (1 + expf(tmp));
    }
}