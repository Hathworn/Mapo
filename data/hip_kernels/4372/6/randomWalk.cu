#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomWalk(double *results, int *crossTimes, double T, int N, int numSims, double lowerThreshold, double upperThreshold, int *dev_failCross, double seconds) {
int crossTimeIndex = threadIdx.x + blockIdx.x * blockDim.x;

if (crossTimeIndex < numSims) {
hiprandState_t state;

hiprand_init (blockIdx.x * 1000 + threadIdx.x + seconds, 0, 0, &state);
double random;
int start = (threadIdx.x + blockIdx.x * blockDim.x) * N;

bool crossed = false;

crossTimes[crossTimeIndex] = 0;
results[start] = 0.0;

for (int j = start + 1; j < start + N; j++) {
random = hiprand_normal_double(&state);
results[j] = results[j-1] + random * sqrt((double) T / N);
if (results[j] >= upperThreshold && !crossed) {
crossTimes[crossTimeIndex] = j - start;
crossed = true;
}
else if (results[j] <= lowerThreshold && !crossed) {
crossTimes[crossTimeIndex] = -1 * (j - start);
crossed = true;
}
}

if (!crossed) {
atomicAdd(dev_failCross, 1);
}


}


/*
Generate 2 doubles at once. Test later to see if this is more efficient:
double hiprand_normal2_double (state);
*/

}