#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomWalk(double *results, int *crossTimes, double T, int N, int numSims, double lowerThreshold, double upperThreshold, int *dev_failCross, double seconds) {
    int crossTimeIndex = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (crossTimeIndex >= numSims) return;  // Early exit if index is out of bounds
    
    hiprandState_t state;
    hiprand_init(blockIdx.x * 1000 + threadIdx.x + seconds, 0, 0, &state);
    
    double random;
    int start = crossTimeIndex * N;  // Use crossTimeIndex instead of recalculating
    bool crossed = false;
    
    crossTimes[crossTimeIndex] = 0;  // Initialize crossTimes array
    results[start] = 0.0;  // Initialize the first result to zero
    
    for (int j = 1; j < N; j++) {  // Simplified loop from 1 to N
        random = hiprand_normal_double(&state);
        results[start + j] = results[start + j - 1] + random * sqrt(T / N);  // Update result directly
        
        if (!crossed) {  // Check crossing condition only if not already crossed
            if (results[start + j] >= upperThreshold) {
                crossTimes[crossTimeIndex] = j;
                crossed = true;
            } else if (results[start + j] <= lowerThreshold) {
                crossTimes[crossTimeIndex] = -j;
                crossed = true;
            }
        }
    }
    
    if (!crossed) atomicAdd(dev_failCross, 1);  // Atomic update for failed crosses
}