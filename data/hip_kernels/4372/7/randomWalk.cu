#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomWalk(double *results, int T, int N) {
hiprandState_t state;
hiprand_init (1234, 0, 0, &state);
double random;

results[0] = 0.0;

for (int j = 1; j < N; j++) {
random = hiprand_normal_double(&state);
results[j] = results[j-1] + random * sqrt((double) T / N);
}

/*
Generate 2 doubles at once. Test later to see if this is more efficient:
double hiprand_normal2_double (state);
*/

}