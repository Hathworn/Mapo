#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomWalk(double *results, int T, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Initialize state for each thread with unique seed
    hiprandState_t state;
    hiprand_init(1234, idx, 0, &state);

    if(idx < N) {
        results[idx] = 0.0;
        
        // Use stride to loop through elements to allow parallelism
        for (int j = idx + 1; j < N; j += blockDim.x * gridDim.x) {
            double random = hiprand_normal_double(&state);
            results[j] = results[j - 1] + random * sqrt((double) T / N);
        }
    }
}