#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void changeValues(double *matrix, int size) {
    // Calculate the unique global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if index is within bounds
    if (index < size) {
        // Multiply by 10 and truncate directly
        matrix[index] = floor(matrix[index] * 10.0);
    }
}