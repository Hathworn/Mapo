#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute input/output plane selection
    int o = blockIdx.x;
    int k = blockIdx.x % input_n;
    gradOutput += o * output_w * output_h;
    gradInput += o * input_w * input_h;

    // Load weight scalar
    float the_weight = weight[k];

    // Pre-calculate thread-specific boundaries
    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Optimize memory access by reusing calculated results; reduce atomic operations
    for (int yy = yy_start; yy < output_h; yy += yy_step)
    {
        for (int xx = xx_start; xx < output_w; xx += xx_step)
        {
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput * the_weight;

            int input_offset = (yy * dH * input_w) + (xx * dW);
            for (int ky = 0; ky < kH; ky++)
            {
                for (int kx = 0; kx < kW; kx++)
                {
                    // Minimize pointer arithmetic inside atomic adds
                    atomicAdd(&gradInput[input_offset + ky * input_w + kx], z);
                }
            }
        }
    }
}