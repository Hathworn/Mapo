#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate offsets to avoid repeated calculations
    int batchOffset = blockIdx.x * input_w * input_n;
    int timeIndex = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    int outputOffset = batchOffset + timeIndex * input_n * dW;
    int gradOutputOffset = blockIdx.x * output_w * input_n + timeIndex * input_n;

    // Use shared memory to store indices and gradOutput for coalesced access
    __shared__ float sharedIndices[TEMPORAL_MAX_POOLING_THREADS];
    __shared__ float sharedGradOutput[TEMPORAL_MAX_POOLING_THREADS];

    int feat = threadIdx.x;
    
    if (timeIndex < output_w) {
        // Load values into shared memory
        sharedIndices[feat] = indices[gradOutputOffset + feat];
        sharedGradOutput[feat] = gradOutput[gradOutputOffset + feat];
        __syncthreads();

        // For all features
        for (feat = 0; feat < input_n; ++feat) {
            // Use shared memory for indices and gradOutput
            atomicAdd(&gradInput[outputOffset + (int)sharedIndices[feat] * input_n + feat], sharedGradOutput[feat]);
        }
        __syncthreads();
    }
}