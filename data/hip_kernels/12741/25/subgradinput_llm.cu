#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinput(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Compute output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;

    // Select input/output plane
    gradOutput = gradOutput + o * output_w * output_h;
    gradInput = gradInput + i * input_w * input_h;

    // Get weight
    float the_weight = weight[k];

    // Load gradOutput into shared memory for coalesced access
    __shared__ float sharedGradOutput[1024]; // adjust size as needed
    int threadIndex = threadIdx.y * blockDim.x + threadIdx.x;
    if (threadIndex < output_w * output_h) {
        sharedGradOutput[threadIndex] = gradOutput[threadIndex] * the_weight;
    }
    __syncthreads();

    // Compute gradInput
    for (int yy = yy_start; yy < output_h; yy += blockDim.y * gridDim.y) {
        for (int xx = xx_start; xx < output_w; xx += blockDim.x) {
            float z = sharedGradOutput[yy * output_w + xx];
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++)
                    ptr_gradInput[kx] += z;
                ptr_gradInput += input_w;
            }
        }
    }
}