#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on block/thread indices
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    float *ptr_output = output + o * output_w * output_h;
    float *ptr_input = input + i * input_w * input_h;

    // Get the bias and weight for the current (k, i)
    float the_weight = weight[k];
    float the_bias = bias[k];

    // Iterate over each output pixel
    for (int yy = yy_start; yy < yy_end; yy += yy_step) {
        for (int xx = xx_start; xx < xx_end; xx += xx_step) {
            // Compute the mean of the input image for the current patch
            float sum = 0;
            float *local_input = ptr_input + yy * dH * input_w + xx * dW;
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    sum += local_input[kx];
                }
                local_input += input_w; // move to next row in the input patch
            }
            // Update the output
            ptr_output[yy * output_w + xx] = the_weight * sum + the_bias;
        }
    }
}