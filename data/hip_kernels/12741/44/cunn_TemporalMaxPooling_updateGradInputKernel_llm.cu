#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate unique thread index for better coalescing
    int time_index = threadIdx.x + blockIdx.y * blockDim.x;
    
    // Skip threads that are out of bounds
    if (time_index >= output_w) return;

    // Compute base pointers
    float *gradInput_data = gradInput + blockIdx.x * input_w * input_n + time_index * input_n * dW;
    float *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + time_index * input_n;
    float *indices_data = indices + blockIdx.x * output_w * input_n + time_index * input_n;
    
    // Unroll loop for efficiency
    for (int feat = 0; feat < input_n; ++feat) {
        int index = (int)indices_data[feat] * input_n + feat;
        atomicAdd(&gradInput_data[index], gradOutput_data[feat]);
    }
}