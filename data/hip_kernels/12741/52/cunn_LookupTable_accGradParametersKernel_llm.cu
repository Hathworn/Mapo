#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_LookupTable_accGradParametersKernel(float *input, float *indices, float *gradOutput, float *gradWeight, float *count, float defaultScale, long numel, long stride, int paddingValue) {

    int idx = blockIdx.x * 4 + threadIdx.y;
    const int SZ = 4; // Number of values processed by each thread (grain size)

    // Ensure idx is within bounds and input conditions are met
    if (idx < numel && (idx == 0 || input[idx] != input[idx - 1]) && input[idx] != paddingValue) {
        do {
            const int startFeature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
            const int weightRow = ((int) input[idx] - 1) * stride;
            const int gradOutputRow = ((int) indices[idx] - 1) * stride;
            const float scale = count ? defaultScale / count[idx] : defaultScale;

            float gradient[SZ];
            float weight[SZ];

            // Load gradient and weight efficiently using unrolling
            #pragma unroll
            for (int ii = 0; ii < SZ; ii++) {
                int featureDim = startFeature + ii * warpSize; // Use warpSize instead of WARP_SIZE for HIP
                if (featureDim < stride) {
                    gradient[ii] = gradOutput[gradOutputRow + featureDim];
                    weight[ii] = gradWeight[weightRow + featureDim];
                }
            }

            // Update weights using unrolling
            #pragma unroll
            for (int ii = 0; ii < SZ; ii++) {
                weight[ii] += gradient[ii] * scale;
            }

            // Write back updated weights efficiently
            #pragma unroll
            for (int ii = 0; ii < SZ; ii++) {
                int featureDim = startFeature + ii * warpSize; // Use warpSize instead of WARP_SIZE for HIP
                if (featureDim < stride) {
                    gradWeight[weightRow + featureDim] = weight[ii];
                }
            }

            idx++;
        } while (idx < numel && input[idx] == input[idx - 1]);
    }
}