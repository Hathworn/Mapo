#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor)
{
    int x, y, z, w;
    w = ii % d3;
    ii = ii/d3;
    z = ii % d2;
    ii = ii/d2;
    y = ii % d1;
    ii = ii/d1;
    x = ii;
    w = w/scale_factor;
    z = z/scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x*d1+y)*d2)+z)*d3+w;
}

__global__ void upscale(float *input, float *output, long no_elements, int scale_factor, int d1, int d2, int d3)
{
    // Calculate global thread index in x and y dimensions
    long idx_x = threadIdx.x + blockIdx.x * blockDim.x;
    long idx_y = threadIdx.y + blockIdx.y * blockDim.y;
    // Calculate the 1D index from the 2D grid
    long ii = idx_x + idx_y * (blockDim.x * gridDim.x);
    if (ii >= no_elements) return; // Boundary check
    int ipidx = translate_idx(ii, d1, d2, d3, scale_factor);
    output[ii] = input[ipidx];
}