#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void atomicadaptivemaxgradinput( float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w )
{
    // Compute global thread indices
    int o = blockIdx.x;
    int i = o;
    
    int xx = blockIdx.z * blockDim.x + threadIdx.x;
    int yy = blockIdx.y * blockDim.y + threadIdx.y;

    if (xx < output_w && yy < output_h) {  // Ensure within bounds
        // Select input/output plane
        gradOutput = gradOutput + o * output_w * output_h;
        gradInput = gradInput + i * input_w * input_h;
        indices_x = indices_x + o * output_w * output_h;
        indices_y = indices_y + o * output_w * output_h;

        int y_start = (int)floor(float(yy) / output_h * input_h);
        int x_start = (int)floor(float(xx) / output_w * input_w);

        float *ptr_gradInput = gradInput + y_start * input_w + x_start;
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float *ptr_ind_x = indices_x + yy * output_w + xx;
        float *ptr_ind_y = indices_y + yy * output_w + xx;
        float z = *ptr_gradOutput;

        int argmax_x = (*ptr_ind_x) - 1;
        int argmax_y = (*ptr_ind_y) - 1;

        // Atomic add since different threads could update same variable
        atomicAdd(&(ptr_gradInput[argmax_x + argmax_y * input_w]), z);
    }
}