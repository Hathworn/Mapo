#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w)
{
    // Compute global thread indices
    int xx = blockIdx.x * blockDim.x + threadIdx.x;
    int yy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds 
    if (xx >= output_w || yy >= output_h) return;

    // Plane offset
    int o = blockIdx.z; // Using blockIdx.z for multiple image planes as each x, y block handles a different plane
    gradOutput += o * output_w * output_h;
    gradInput += o * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // Calculate input indices
    int y_start = __float2int_rd(float(yy) / output_h * input_h);
    int x_start = __float2int_rd(float(xx) / output_w * input_w);

    // Compute addresses
    float *ptr_gradInput = gradInput + y_start * input_w + x_start;
    float *ptr_gradOutput = gradOutput + yy * output_w + xx;
    float *ptr_ind_x = indices_x + yy * output_w + xx;
    float *ptr_ind_y = indices_y + yy * output_w + xx;
    
    // Update gradInput with gradient from gradOutput
    int argmax_x = __float2int_rd(*ptr_ind_x) - 1;
    int argmax_y = __float2int_rd(*ptr_ind_y) - 1;

    atomicAdd(&ptr_gradInput[argmax_x + argmax_y * input_w], *ptr_gradOutput);
}