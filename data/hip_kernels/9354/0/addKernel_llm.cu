#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

// Improved global kernel function
__global__ void addKernel(double *c, const double *a, const double *b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < N) { // Ensure thread index is within array bounds
        c[i] = a[i] + b[i];
    }
}