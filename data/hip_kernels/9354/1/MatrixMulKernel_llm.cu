#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void MatrixMulKernel(double *OutMat, double *Mat1, double *Mat2, int Arows, int Acols, int Bcols) {
    // Obtain row and column indices
    int blockrow = blockIdx.y;
    int row = threadIdx.y;
    int blockcol = blockIdx.x;
    int col = threadIdx.x;

    // Declare shared memory
    __shared__ double subAshared[BLOCKSIZE][BLOCKSIZE];
    __shared__ double subBshared[BLOCKSIZE][BLOCKSIZE];
    double Cvalue = 0;

    // Improved loop conditions
    for (int B = 0; B < (Acols + BLOCKSIZE - 1) / BLOCKSIZE; B++) { 
        // Calculate indices
        int Mat1index = min(row + blockrow * BLOCKSIZE, Arows - 1) * Acols + col + B * BLOCKSIZE;
        int Mat2index = min(B * BLOCKSIZE + row, Acols - 1) * Bcols + blockcol * BLOCKSIZE + col;

        // Load data to shared memory with boundary check optimizations
        subAshared[row][col] = (Mat1index < Arows * Acols) ? Mat1[Mat1index] : 0.0;
        subBshared[row][col] = (Mat2index < Acols * Bcols) ? Mat2[Mat2index] : 0.0;

        __syncthreads();

        // Compute using shared memory
        for (int j = 0; j < BLOCKSIZE; j++) {
            Cvalue += subAshared[row][j] * subBshared[j][col];
        }

        __syncthreads();
    }
    // Store the result
    if (blockrow * BLOCKSIZE + row < Arows && blockcol * BLOCKSIZE + col < Bcols) {
        OutMat[(blockrow * BLOCKSIZE + row) * Bcols + (blockcol * BLOCKSIZE + col)] = Cvalue;
    }
}