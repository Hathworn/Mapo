#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Use blockDim.x for grid-stride loop to process larger arrays
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (; i < size; i += stride) {
        c[i] = a[i] + b[i];
    }
}