#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void zero_dm_outliers_kernel_two(unsigned short *d_input, int nchans, int nsamp) {
    // Initialize shared memory for block-level mean and standard deviation
    __shared__ float g_mean[1024];
    __shared__ float g_stdev[1024];

    int c = blockIdx.x * blockDim.x + threadIdx.x;

    // Local variables for mean, standard deviation, and related computations
    float mean = MEAN;
    float mean_last = 0.0f;
    float stdev = 1000000.0f;
    float sum = 0.0f;
    float sum_squares = 0.0f;
    int count = 0, iters = 0;
    float cutoff = (CUT * stdev);

    // Iterative process for calculating mean and standard deviation
    while (fabsf(mean - mean_last) > ACC) {  // Use fabsf for floating point absolute value
        sum = 0.0f;
        sum_squares = 0.0f;
        count = 0;
        
        for (int t = 0; t < nsamp; t++) {
            float data = static_cast<float>(d_input[t * nchans + c]);
            if (data < (mean + cutoff) && data > (mean - cutoff)) {
                sum += data;
                sum_squares += (data * data);
                count++;
            }
        }
        
        mean_last = mean;
        mean = (sum / static_cast<float>(count));
        sum_squares = ((sum_squares / count) - (mean * mean));
        stdev = sqrtf(sum_squares);  // Use sqrtf for floating point square root
        cutoff = (CUT * stdev);
        iters++;
        if (iters > ITER) break;
    }

    // Handle cases where standard deviation computation failed or reached iteration limit
    if (count == 0 || iters > ITER || mean == 0.0f || stdev == 0.0f) {
        for (int t = 0; t < nsamp; t++) {
            d_input[t * nchans + c] = MEAN;
        }
        g_mean[threadIdx.x] = mean = MEAN;
        g_stdev[threadIdx.x] = stdev = 0.0f;
    } else {
        g_mean[threadIdx.x] = mean;
        g_stdev[threadIdx.x] = stdev;
    }

    __syncthreads();

    // Block-level calculation of mean of means and mean of standard deviations for further filtering
    float mean_of_mean = 0.0f, stdev_of_mean = 0.0f, m_cutoff = 0.0f;
    float mean_of_stdev = 0.0f, stdev_of_stdev = 0.0f, s_cutoff = 0.0f;
    sum_squares = 0.0f;

    for (int i = 0; i < blockDim.x; i++) {
        mean_of_mean += g_mean[i];
        sum_squares += (g_mean[i] * g_mean[i]);
    }

    mean_of_mean /= blockDim.x;
    sum_squares = ((sum_squares / blockDim.x) - (mean_of_mean * mean_of_mean));
    stdev_of_mean = sqrtf(sum_squares);
    m_cutoff = (3.0f * stdev_of_mean);

    sum_squares = 0.0f;
    for (int i = 0; i < blockDim.x; i++) {
        mean_of_stdev += g_stdev[i];
        sum_squares += (g_stdev[i] * g_stdev[i]);
    }

    mean_of_stdev /= blockDim.x;
    sum_squares = ((sum_squares / blockDim.x) - (mean_of_stdev * mean_of_stdev));
    stdev_of_stdev = sqrtf(sum_squares);
    s_cutoff = (3.0f * stdev_of_stdev);

    // Final filtering based on mean and standard deviation thresholds
    if (fabsf(g_mean[threadIdx.x] - mean_of_mean) > m_cutoff || fabsf(g_stdev[threadIdx.x] - mean_of_stdev) > s_cutoff) {
        for (int t = 0; t < nsamp; t++) {
            d_input[t * nchans + c] = MEAN;
        }
    } else {
        for (int t = 0; t < nsamp; t++) {
            float data = static_cast<float>(d_input[t * nchans + c]);
            if ((data - mean < R_CUT * stdev) && (data - mean > -R_CUT * stdev)) {
                d_input[t * nchans + c] = static_cast<unsigned short>(data - mean + MEAN);
            } else {
                d_input[t * nchans + c] = MEAN;
            }
        }
    }
}