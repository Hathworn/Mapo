#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SNR_GPU_limited(float *d_FIR_input, float *d_SNR_output, ushort *d_SNR_taps, float *d_MSD, int x_steps, int nTaps, int nColumns, int offset) {
    int local_id = threadIdx.x & ( WARP - 1 );
    int warp_id = threadIdx.x >> 5;
    int dim_y = blockDim.x >> 5;
    
    int pos_x, pos_y;
    float old_SNR, new_SNR;
    
    float signal_mean = d_MSD[0];
    float signal_sd = d_MSD[1];
    
    pos_y = ( blockIdx.y * dim_y + warp_id ) * nColumns;
    pos_x = blockIdx.x * WARP * x_steps + local_id;

    // Precompute some common values outside of the loop to reduce computation inside the loop
    float base_sd_factor = signal_sd;
    int col_limit = nColumns - offset;

    for (int xf = 0; xf < x_steps; xf++) {
        if (pos_x < col_limit) {
            old_SNR = d_SNR_output[pos_y + pos_x];
            new_SNR = (d_FIR_input[pos_y + pos_x] - signal_mean) / base_sd_factor;
            if (nTaps == 1 || new_SNR > old_SNR) {  // Simplified conditional logic for nTaps
                d_SNR_output[pos_y + pos_x] = new_SNR;
                d_SNR_taps[pos_y + pos_x] = nTaps;
            }
        } else if (pos_x >= col_limit && pos_x < nColumns) {
            d_SNR_output[pos_y + pos_x] = 0;
            d_SNR_taps[pos_y + pos_x] = 0;
        }
        pos_x += WARP;  // Simplified position increment
    }
}