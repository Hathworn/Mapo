#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PD_ZC_GPU_KERNEL(float *d_input, float *d_output, int maxTaps, int nTimesamples, int nLoops)
{
    int x_r, y_r, x_w, y_w;
    int Elements_per_block = PD_NTHREADS * PD_NWINDOWS;

    y_r = (blockIdx.y * blockDim.y + threadIdx.y) * nTimesamples;
    x_r = (blockIdx.x + 1) * Elements_per_block + threadIdx.x;

    y_w = (blockIdx.y * blockDim.y + threadIdx.y) * (maxTaps - 1) * gridDim.x;
    x_w = blockIdx.x * (maxTaps - 1) + threadIdx.x;

    // Unrolling the loop for better performance
    #pragma unroll
    for (int f = 0; f < nLoops; f++)
    {
        if (x_r < nTimesamples && threadIdx.x < (maxTaps - 1))
        {
            // Removing loop-dependent variables calculation from loop
            int output_idx = x_w + y_w + f * WARP;
            int input_idx = x_r + y_r + f * WARP;
            d_output[output_idx] = d_input[input_idx];
        }
    }
}