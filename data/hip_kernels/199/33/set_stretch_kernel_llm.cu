#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_stretch_kernel(int samps, float mean, float *d_input) {
    // Calculate global thread index
    int t = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within the valid range
    if (t < samps) {
        d_input[t] = mean; // Set value at the valid index
    }
}