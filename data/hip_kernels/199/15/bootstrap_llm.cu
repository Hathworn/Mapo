#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bootstrap(int bins, int num_els, int num_boots, float *g_idata, double *g_odata, unsigned int *g_irand)
{
    float myResample = 0.0f;

    // Precompute constants outside the loop
    unsigned int constant = (4294967295 / (bins - blockDim.x));
    int constant2 = blockIdx.x * bins;
    int dmid = bins * (blockDim.y * blockIdx.y + threadIdx.y);
    int thread_offset = dmid + threadIdx.x;

    // Efficiently iterate over bins and accumulate result
    for (int i = 0; i < bins; i++)
    {
        int rid = (g_irand[constant2 + i] / constant);
        myResample += g_idata[rid + thread_offset];
    }

    dmid = num_boots * (blockDim.y * blockIdx.y + threadIdx.y);
    g_odata[dmid + threadIdx.x + blockDim.x * blockIdx.x] = ((double)myResample / (double)num_els);
}