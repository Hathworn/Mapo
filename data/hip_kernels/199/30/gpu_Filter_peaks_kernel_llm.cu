#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_Filter_peaks_kernel(unsigned int *d_new_peak_list_DM, unsigned int *d_new_peak_list_TS, unsigned int *d_new_peak_list_BW, float *d_new_peak_list_SNR, unsigned int *d_peak_list_DM, unsigned int *d_peak_list_TS, unsigned int *d_peak_list_BW, float *d_peak_list_SNR, unsigned int nElements, unsigned int max_distance, int nLoops, int max_list_pos, int *gmem_pos){
// PPF_DPB = 128 //this is because I set nThreads to 64
// PPF_PEAKS_PER_BLOCK = something small like 10
__shared__ float s_data_snr[PPF_DPB];
__shared__ int s_data_dm[PPF_DPB];
__shared__ int s_data_ts[PPF_DPB];
__shared__ int s_flag[PPF_NTHREADS];
int elements_pos, pos;
float snr, distance, fs, fd;

// Initialize flags
if (threadIdx.x < PPF_PEAKS_PER_BLOCK) {
    s_flag[threadIdx.x] = 1;
} else {
    s_flag[threadIdx.x] = 0;
}

for (int f = 0; f < nLoops; f++) {
    // Load new data blob
    pos = PPF_DPB * f + threadIdx.x;
    if (pos < nElements) {
        s_data_dm[threadIdx.x] = d_peak_list_DM[pos];
        s_data_ts[threadIdx.x] = d_peak_list_TS[pos];
        s_data_snr[threadIdx.x] = d_peak_list_SNR[pos];
    } else {
        // Use sentinel values for inactive threads
        s_data_dm[threadIdx.x] = 0;
        s_data_ts[threadIdx.x] = 0;
        s_data_snr[threadIdx.x] = -1000;
    }

    pos = PPF_DPB * f + threadIdx.x + PPF_NTHREADS;
    if (pos < nElements) {
        s_data_dm[threadIdx.x + PPF_NTHREADS] = d_peak_list_DM[pos];
        s_data_ts[threadIdx.x + PPF_NTHREADS] = d_peak_list_TS[pos];
        s_data_snr[threadIdx.x + PPF_NTHREADS] = d_peak_list_SNR[pos];
    } else {
        s_data_dm[threadIdx.x + PPF_NTHREADS] = 0;
        s_data_ts[threadIdx.x + PPF_NTHREADS] = 0;
        s_data_snr[threadIdx.x + PPF_NTHREADS] = -1000;
    }

    __syncthreads();

    elements_pos = blockIdx.x * PPF_PEAKS_PER_BLOCK;
    for (int p = 0; p < PPF_PEAKS_PER_BLOCK; p++) {
        if ((s_flag[p]) && ((elements_pos + p) < nElements)) {
            int d = d_peak_list_DM[elements_pos + p]; // DM
            int s = d_peak_list_TS[elements_pos + p]; // Time
            snr = d_peak_list_SNR[elements_pos + p]; // SNR

            // Compare with first element
            if (s_data_snr[threadIdx.x] >= snr) {
                fs = (float)s_data_dm[threadIdx.x] - (float)d;
                fd = (float)s_data_ts[threadIdx.x] - (float)s;
                distance = (fd * fd + fs * fs);
                if ((distance < (float)max_distance) && (distance != 0)) {
                    s_flag[p] = 0;
                }
            }

            // Compare with second element
            if (s_data_snr[threadIdx.x + PPF_NTHREADS] >= snr) {
                fs = (float)s_data_dm[threadIdx.x + PPF_NTHREADS] - (float)d;
                fd = (float)s_data_ts[threadIdx.x + PPF_NTHREADS] - (float)s;
                distance = (fd * fd + fs * fs);
                if ((distance < (float)max_distance) && (distance != 0)) {
                    s_flag[p] = 0;
                }
            }
        }
    } // for p
}

// Saving peaks that got through
elements_pos = blockIdx.x * PPF_PEAKS_PER_BLOCK;
if (threadIdx.x < PPF_PEAKS_PER_BLOCK) {
    if ((s_flag[threadIdx.x] == 1) && ((elements_pos + threadIdx.x) < nElements)) {
        int list_pos = atomicAdd(gmem_pos, 1);
        if (list_pos < max_list_pos) {
            d_new_peak_list_DM[list_pos] = d_peak_list_DM[elements_pos + threadIdx.x];
            d_new_peak_list_TS[list_pos] = d_peak_list_TS[elements_pos + threadIdx.x];
            d_new_peak_list_BW[list_pos] = d_peak_list_BW[elements_pos + threadIdx.x];
            d_new_peak_list_SNR[list_pos] = d_peak_list_SNR[elements_pos + threadIdx.x];
        }
    }
}
}