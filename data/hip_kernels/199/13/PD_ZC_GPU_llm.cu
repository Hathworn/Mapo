#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PD_ZC_GPU(float *d_input, float *d_output, int maxTaps, int nTimesamples, int nLoops) {
    int x_r, y_r, x_w, y_w;
    int Elements_per_block = PD_NTHREADS * PD_NWINDOWS;

    // Calculate read and write indices
    y_r = (blockIdx.y * blockDim.y + threadIdx.y) * nTimesamples;
    x_r = (blockIdx.x + 1) * Elements_per_block + threadIdx.x;

    y_w = (blockIdx.y * blockDim.y + threadIdx.y) * (maxTaps - 1) * gridDim.x;
    x_w = blockIdx.x * (maxTaps - 1) + threadIdx.x;

    // Added shared memory for optimized memory access
    __shared__ float sharedMem[PD_NTHREADS];

    // Optimize loop to access data in shared memory
    for (int f = 0; f < nLoops; f++) {
        if (x_r < nTimesamples && threadIdx.x < (maxTaps - 1)) {
            sharedMem[threadIdx.x] = d_input[x_r + y_r + f * WARP];  // Load to shared memory
            __syncthreads();  // Synchronize threads within block
            d_output[x_w + y_w + f * WARP] = sharedMem[threadIdx.x]; // Write from shared memory
        }
    }
}