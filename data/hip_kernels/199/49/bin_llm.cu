#include "hip/hip_runtime.h"
```cuda
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bin(unsigned short *d_input, float *d_output, int in_nsamp) {

    // Calculate output and input indices for reading/writing
    int c = blockIdx.y * BINDIVINF + threadIdx.y;
    int t_out = blockIdx.x * BINDIVINT + threadIdx.x;
    int t_in = 2 * t_out;

    // Pre-calculate output sample size
    int out_nsamp = in_nsamp / 2;

    // Calculate flat memory indices for output and input
    size_t shift_one = c * out_nsamp + t_out;
    size_t shift_two = c * in_nsamp + t_in;

    // Load input data, compute the average, store to output
    d_output[shift_one] = (float)(d_input[shift_two] + d_input[shift_two + 1]) * 0.5f;

}