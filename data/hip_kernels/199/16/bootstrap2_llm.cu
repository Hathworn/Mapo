#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bootstrap2(int bins, int num_els, int num_boots, float *g_idata, double *g_odata, unsigned int *g_irand)
{
    // Initialize resample variable
    float myResample = 0.0f;

    // Precompute some values outside the loop
    int constant = (4294967295 / bins);
    int constant2 = blockIdx.x * bins;
    int dmid = bins * (blockDim.y * blockIdx.y + threadIdx.y);
    int threadOffset = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Loop through bins
    for (int i = 0; i < bins; i++)
    {
        // Calculate random index efficiently
        int rid = g_irand[constant2 + i] / constant;

        // Accumulate resampling data
        myResample += g_idata[dmid + ((rid + threadIdx.x) % bins)];
    }

    // Store the result in global memory
    dmid = num_boots * (blockDim.y * blockIdx.y + threadIdx.y);
    g_odata[dmid + threadOffset] = ((double)myResample / (double)num_els);
}