#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MSD_GPU_Interpolate_linear(float *d_MSD_DIT, float *d_MSD_interpolated, int *d_MSD_DIT_widths, int MSD_DIT_size, int *boxcar, int max_width_performed) {
    int tid = threadIdx.x;
    if (boxcar[tid] <= max_width_performed) {
        int desired_width = boxcar[tid];
        int position = (int)floorf(log2f((float)desired_width));

        float width1 = d_MSD_DIT_widths[position];
        float mean1 = d_MSD_DIT[position * MSD_RESULTS_SIZE];
        float StDev1 = d_MSD_DIT[position * MSD_RESULTS_SIZE + 1];

        // Efficient condition check and memory access
        if (position == MSD_DIT_size - 1 && width1 == (int)desired_width) {
            d_MSD_interpolated[tid * 2] = mean1;
            d_MSD_interpolated[tid * 2 + 1] = StDev1;
        } else {
            float width2 = d_MSD_DIT_widths[position + 1];
            float distance_in_width = width2 - width1;

            float mean2 = d_MSD_DIT[(position + 1) * MSD_RESULTS_SIZE];
            float distance_in_mean = mean2 - mean1;

            float StDev2 = d_MSD_DIT[(position + 1) * MSD_RESULTS_SIZE + 1];
            float distance_in_StDev = StDev2 - StDev1;

            // Precalculate coefficients to reduce division operations
            float coeff = ((float)desired_width - width1) / distance_in_width;
            d_MSD_interpolated[tid * 2] = mean1 + distance_in_mean * coeff;
            d_MSD_interpolated[tid * 2 + 1] = StDev1 + distance_in_StDev * coeff;
        }
    }
}