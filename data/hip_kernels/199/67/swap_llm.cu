#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void swap(unsigned short *d_input, float *d_output, int nchans, int nsamp) {
    
    // Calculate global thread index
    size_t t = blockIdx.x * blockDim.x + threadIdx.x;
    size_t c = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds to prevent out-of-bounds access
    if (t < nsamp && c < nchans) {
        d_input[c * nsamp + t] = (unsigned short) __ldg(&d_output[c * nsamp + t]);
    }
}