#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_corner_turn_kernel(float *d_input, float *d_output, int primary_size, int secondary_size) {
    // Calculate indices using threadIdx and blockIdx
    size_t primary = blockIdx.x * blockDim.x + threadIdx.x;
    size_t secondary = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds before accessing memory
    if (primary < primary_size && secondary < secondary_size) {
        // Use __ldg for efficient memory access
        d_output[primary * secondary_size + secondary] = __ldg(&d_input[secondary * primary_size + primary]);
    }
}