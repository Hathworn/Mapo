#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stats_kernel(int half_samps, float *d_sum, float *d_sum_square, float *d_signal_power)
{
    // Use shared memory to reduce global memory access
    __shared__ float shared_sum[256];
    __shared__ float shared_sum_square[256];
    
    int t = blockIdx.x * blockDim.x * STATSLOOP + threadIdx.x;
    int idx = threadIdx.x;

    float local = 0.0;
    float sum = 0.0;
    float sum_square = 0.0;

    // Loop unrolling and use of shared memory for better performance
    for (int i = 0; i < STATSLOOP; i++)
    {
        local = d_signal_power[t + i * blockDim.x];
        sum += local;
        sum_square += local * local;
    }
    
    // Store partial sums in shared memory
    shared_sum[idx] = sum;
    shared_sum_square[idx] = sum_square;

    __syncthreads();  // Ensure all threads have written to shared memory

    // Parallel reduction within block for final sum and sum_square
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) 
    {
        if (idx < stride) 
        {
            shared_sum[idx] += shared_sum[idx + stride];
            shared_sum_square[idx] += shared_sum_square[idx + stride];
        }
        __syncthreads();
    }

    // Write the result of reduction for each block to global memory
    if (idx == 0)
    {
        d_sum[blockIdx.x] = shared_sum[0];
        d_sum_square[blockIdx.x] = shared_sum_square[0];
    }
}