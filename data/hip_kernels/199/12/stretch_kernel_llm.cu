#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimized for better memory access and reduced computation in loop
__global__ void stretch_kernel(int acc, int samps, float tsamp, float *d_input, float *d_output, float t_zero, float multiplier, float tsamp_inverse) {
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Precompute common terms to save computation inside the loop
    float multiplier_offset = multiplier * (t - 1.0f);
    float p_time = t * (t_zero + multiplier_offset);
    
    int stretch_index = __float2int_rz(p_time * tsamp_inverse);

    // Ensure the index is within bounds before writing to global memory
    if (stretch_index >= 0 && stretch_index < samps) {
        d_output[stretch_index] = d_input[t];
    }
}