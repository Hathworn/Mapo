#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GPU_simple_power_and_interbin_kernel(float2 *d_input_complex, float *d_output_power, float *d_output_interbinning, int nTimesamples, float norm){
    int pos_x = blockIdx.x * blockDim.x + threadIdx.x;
    int pos_y = blockIdx.y * ((nTimesamples >> 1) + 1);

    // Initialize A and B to avoid using uninitialized data
    float2 A = {0.0f, 0.0f};
    float2 B = {0.0f, 0.0f};

    if ((pos_x < (nTimesamples >> 1)) && (pos_x > 0)) {
        A = d_input_complex[pos_y + pos_x];
        B = d_input_complex[pos_y + pos_x + 1];

        // Use reciprocal multiplication for normalization for performance
        float inv_norm = 1.0f / norm;
        A.x *= inv_norm;
        A.y *= inv_norm;
        B.x *= inv_norm;
        B.y *= inv_norm;
    }

    if (pos_x < (nTimesamples >> 1)) {
        // Calculate power directly without storing intermediate values
        float power = A.x * A.x + A.y * A.y;
        d_output_power[blockIdx.y * (nTimesamples >> 1) + pos_x] = power;
        d_output_interbinning[blockIdx.y * nTimesamples + 2 * pos_x] = power;
        d_output_interbinning[blockIdx.y * nTimesamples + 2 * pos_x + 1] = 0.616850275f * ((A.x - B.x) * (A.x - B.x) + (A.y - B.y) * (A.y - B.y));
    }
}