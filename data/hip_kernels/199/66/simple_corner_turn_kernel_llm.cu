#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simple_corner_turn_kernel(unsigned short *d_input, float *d_output, int nchans, int nsamp) {
    // Use a single index to improve coalesced memory access
    size_t idx = blockIdx.y * blockDim.y * nsamp + blockIdx.x * blockDim.x + threadIdx.y * nsamp + threadIdx.x;

    if (threadIdx.y < nchans && threadIdx.x < nsamp) {
        // Using shared memory could be unnecessary without further context
        d_output[idx] = (float) __ldg(&d_input[idx]);
    }
}