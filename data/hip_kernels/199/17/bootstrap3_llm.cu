#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bootstrap3(int bins, int num_els, int num_boots, float *g_idata, double *g_odata, unsigned int *g_irand)
{
    // Use registers for frequently used variables to reduce global memory access time
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int dmid_base = bins * (blockDim.y * blockIdx.y + threadIdx.y);
    int dmid_output = num_boots * (blockDim.y * blockIdx.y + threadIdx.y);

    // Initialize myResample to 0.0 to prevent using garbage value
    float myResample = 0.0f;
    int constant = (4294967295 / bins);

    // Loop unrolling if it's known that 'bins' won't change frequently
    for (int i = 0; i < bins; i++)
    {
        int rid = g_irand[id * bins + i] / constant;  // Improved access pattern
        myResample += g_idata[dmid_base + rid];
    }

    // Write final result to global memory with computed index
    g_odata[dmid_output + id] = ((double)myResample / (double)num_els);
}