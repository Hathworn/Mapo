#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiplication(int * A,int * B,int * C,int N){
    // Calculate the row and column index of the element
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check boundaries within matrix dimensions
    if (row < N && col < N) {
        int sum = 0;
        
        // Use shared memory for sub-matrix tiles
        __shared__ int shared_A[32][32];
        __shared__ int shared_B[32][32];
        
        // Iterate through sub-matrix tiles
        for (int t = 0; t < (N + 31) / 32; ++t) {
            // Load data into shared memory
            if (row < N && t * 32 + threadIdx.x < N) {
                shared_A[threadIdx.y][threadIdx.x] = A[row * N + t * 32 + threadIdx.x];
            } else {
                shared_A[threadIdx.y][threadIdx.x] = 0;
            }
            
            if (t * 32 + threadIdx.y < N && col < N) {
                shared_B[threadIdx.y][threadIdx.x] = B[(t * 32 + threadIdx.y) * N + col];
            } else {
                shared_B[threadIdx.y][threadIdx.x] = 0;
            }
            
            __syncthreads();
            
            // Compute the sum for this tile
            for (int i = 0; i < 32; ++i) {
                sum += shared_A[threadIdx.y][i] * shared_B[i][threadIdx.x];
            }
            
            __syncthreads();
        }
        
        // Write the result to the matrix C
        C[row * N + col] = sum;
    }
}