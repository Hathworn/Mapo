#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiplication(int * A, int * B, int * C, int N, int M, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < K) { // Check bounds
        int sum = 0;
        for (int i = 0; i < M; i++) {
            sum += A[row * M + i] * B[i * K + col]; // Correct index calculation for matrix multiplication
        }
        C[row * K + col] = sum; // Correct index calculation for result matrix
    }
}