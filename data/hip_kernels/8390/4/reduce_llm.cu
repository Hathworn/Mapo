#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce(int * vector, int size, int pot) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = pot / 2;

    // Use a loop unrolling optimization strategy
    for (; stride > 0; stride >>= 1) {
        if (idx < stride && idx + stride < size) {
            vector[idx] += vector[idx + stride];
        }
        __syncthreads();
    }
}