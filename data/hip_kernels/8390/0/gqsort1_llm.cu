#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define MAXR(sz) (((sz)+MAXSEQ-1)/MAXSEQ+1)
#define MAXT MAXR(MAXN)
int MAXN;
int MAXSEQ;
int THRN;

typedef struct secuence{
    int start,end,pivot;
} secuence;

typedef struct block{
    secuence seq,parent;
    int blockcount,id,bid;
} block;

__global__ void gqsort1(block * blocks, int * d, int * LT, int * GT) {
    // Calculate unique thread id across all blocks
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;

    int gt = 0, lt = 0;
    int pivot, start, end;

    if (threadIdx.x == 0) {
        pivot = blocks[blockIdx.x].seq.pivot;
        start = blocks[blockIdx.x].seq.start;
        end = blocks[blockIdx.x].seq.end;
        LT[blockIdx.x] = 0;
        GT[blockIdx.x] = 0;
    }

    __syncthreads();

    // Distribute the workload evenly across all threads
    int index = start + global_id;
    while (index < end) {
        if (d[index] < pivot) {
            lt++;
        } else if (d[index] > pivot) {
            gt++;
        }
        index += total_threads;  // Move to the next work item for this thread
    }

    // Use atomic operations to avoid race conditions
    atomicAdd(&LT[blockIdx.x], lt);
    atomicAdd(&GT[blockIdx.x], gt);
}