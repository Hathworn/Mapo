#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define MAXR(sz) (((sz)+MAXSEQ-1)/MAXSEQ+1)
#define MAXT MAXR(MAXN)
int MAXN;
int MAXSEQ;
int THRN;

//===Definicion de estructuras y funciones utiles===

typedef struct secuence{
    int start,end,pivot;
}secuence;

typedef struct block{
    secuence seq,parent;
    int blockcount,id,bid;
}block;

__global__ void gqsort3(block * blocks, int * d, int * _d) {
    int id = blockIdx.x, th = threadIdx.x, cth = blockDim.x;
    int start = blocks[id].seq.start, end = blocks[id].seq.end;

    // Use shared memory to improve performance
    __shared__ int sdata[1024]; // Assuming a max of 1024 threads per block
    if (th < (end - start)) {
        sdata[th] = _d[start + th]; // Load data to shared memory
    }
    __syncthreads();

    for (int j = start + th; j < end; j += cth) {
        // Access shared memory instead of global memory
        d[j] = sdata[j - start];
    }

    return;
}