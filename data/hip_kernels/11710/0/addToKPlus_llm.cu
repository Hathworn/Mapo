#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addToKPlus(int msize, double* a,  double* b, double* c, double* d)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (tid < msize) {
        d[tid] = a[tid] + b[tid] + c[tid]; // Perform addition efficiently
    }
}