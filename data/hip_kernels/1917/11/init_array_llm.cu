#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Cache the factor value to reduce global memory access
    int local_factor = *factor;

    // Loop unrolling for improved performance
    for (int i = 0; i<num_iterations; i+=4) {
        g_data[idx] += local_factor;
        if (i+1 < num_iterations) g_data[idx] += local_factor;
        if (i+2 < num_iterations) g_data[idx] += local_factor;
        if (i+3 < num_iterations) g_data[idx] += local_factor;
    }
}