#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateVelocity_k(float2 *v, float *vx, float *vy, int dx, int pdx, int dy, int lb, size_t pitch) {
    // Calculate global thread index for x and y directions
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    float scale = 1.f / (dx * dy); // Precompute scale factor once

    // Check bounds in the x direction
    if (gtidx < dx) {
        for (int p = 0; p < lb; p++) {
            int fi = gtidy + p; // Calculate the y position in the global domain
            // Check bounds in the y direction
            if (fi < dy) {
                int fjr = fi * pdx + gtidx; // Flattened index for vx and vy
                float vxterm = vx[fjr];
                float vyterm = vy[fjr];

                // Populate the new velocity term after scaling
                float2 nvterm;
                nvterm.x = vxterm * scale;
                nvterm.y = vyterm * scale;

                // Calculate pointer offset using pitch and transfer the normalized value
                float2 *fj = (float2*)((char*)v + fi * pitch) + gtidx;
                *fj = nvterm;
            }
        }
    }
}