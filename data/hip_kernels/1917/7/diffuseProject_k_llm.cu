#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void diffuseProject_k(float2 *vx, float2 *vy, int dx, int dy, float dt, float visc, int lb) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    
    // Check if gtidx is within bounds once
    if (gtidx < dx) {
        for (int p = 0; p < lb; p++) {
            // Compute fi and check within bounds
            int fi = gtidy + p;
            if (fi < dy) {
                int fj = fi * dx + gtidx;

                float2 xterm = vx[fj];
                float2 yterm = vy[fj];

                int iix = gtidx;
                int iiy = (fi > dy / 2) ? (fi - dy) : fi;

                // Precompute common factors
                float kk = static_cast<float>(iix * iix + iiy * iiy); // k^2
                float diff = 1.f / (1.f + visc * dt * kk);

                // Apply diffusion to velocity components
                xterm.x *= diff; xterm.y *= diff;
                yterm.x *= diff; yterm.y *= diff;

                // Projection logic if kk is greater than 0
                if (kk > 0.f) {
                    float rkk = 1.f / kk;
                    float rkp = (iix * xterm.x + iiy * yterm.x);
                    float ikp = (iix * xterm.y + iiy * yterm.y);

                    // Simplified projection calculations
                    float iix_rkk = iix * rkk;
                    float iiy_rkk = iiy * rkk;

                    xterm.x -= iix_rkk * rkp;
                    xterm.y -= iix_rkk * ikp;
                    yterm.x -= iiy_rkk * rkp;
                    yterm.y -= iiy_rkk * ikp;
                }

                vx[fj] = xterm;
                vy[fj] = yterm;
            }
        }
    }
}