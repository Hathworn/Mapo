#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(int *array, int arrayCount)
{
    // Calculate unique thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Check if within bounds before operation
    if (idx < arrayCount) {
        int value = array[idx];
        array[idx] = value * value;
    }
}