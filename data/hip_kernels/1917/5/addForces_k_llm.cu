#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addForces_k(float2 *v, int dx, int dy, int spx, int spy, float fx, float fy, int r, size_t pitch) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Calculate pointer to v based on tx, ty offsets
    float2 *fj = (float2*)((char*)v + (ty + spy) * pitch) + tx + spx;

    // Use registers for frequently accessed variables
    float2 vterm = *fj;
    int offsetTx = tx - r;
    int offsetTy = ty - r;
    
    // Pre-calculate offset squares
    float tx2 = offsetTx * offsetTx;
    float ty2 = offsetTy * offsetTy;
    
    // Calculate factor with pre-computed squares
    float s = 1.f / (1.f + tx2 * tx2 + ty2 * ty2);

    // Update vterm with computed values
    vterm.x += s * fx;
    vterm.y += s * fy;
    
    // Store result back
    *fj = vterm;
}