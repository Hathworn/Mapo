#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function for vector addition
__global__ void VecAdd(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int N)
{
    // Calculate the index for this thread
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Perform operation only if index is within bounds
    if (i < N)
    {
        // Load values from global to registers for faster access
        float a = A[i];
        float b = B[i];
        
        // Compute the result and store back to global memory
        C[i] = a + b;
    }
}