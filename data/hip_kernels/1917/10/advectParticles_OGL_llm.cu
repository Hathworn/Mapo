#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void advectParticles_OGL(float2 *part, float2 *v, int dx, int dy, float dt, int lb, size_t pitch) {
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    // Use shared memory to reduce global memory access
    __shared__ float2 shared_v[BLOCK_SIZE_Y][BLOCK_SIZE_X]; 

    float2 pterm, vterm;
    
    if (gtidx < dx) {
        for (p = 0; p < lb; p++) {
            int fi = gtidy + p;
            if (fi < dy) {
                int fj = fi * dx + gtidx;
                pterm = part[fj];

                int xvi = ((int)(pterm.x * dx));
                int yvi = ((int)(pterm.y * dy));

                // Load velocity data into shared memory
                if (threadIdx.y < BLOCK_SIZE_Y && threadIdx.x < BLOCK_SIZE_X) {
                    shared_v[threadIdx.y][threadIdx.x] = *((float2*)((char*)v + yvi * pitch) + xvi);
                }
                __syncthreads(); // Ensure all threads have loaded the data

                // Use shared memory to compute new positions
                vterm = shared_v[threadIdx.y][threadIdx.x];
                
                pterm.x += dt * vterm.x;
                pterm.x = pterm.x - (int)pterm.x + (pterm.x < 0.f ? 1.f : 0.f);
                pterm.y += dt * vterm.y;
                pterm.y = pterm.y - (int)pterm.y + (pterm.y < 0.f ? 1.f : 0.f);

                part[fj] = pterm;
            }
        }
    }
}