#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" void check_cuda(hipError_t result, char const *const func, const char *const file, int const line);

__global__ void render_init_kernel(int max_x, int max_y, hiprandState *rand_state) {
    // Cache the block and thread indices
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate a linear index based on the 2D indices
    int pixel_index = iy * max_x + ix;

    // Return early if out of bounds
    if (ix >= max_x || iy >= max_y) return;

    // Initialize the random state without offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}