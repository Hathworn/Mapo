#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cpy(float *a, float *b, int n) {
    // Use shared memory to reduce global memory access
    extern __shared__ float sharedData[];
    
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tid = threadIdx.x;
    
    // Load data into shared memory
    if (i < n) {
        sharedData[tid] = b[i];
    }
    __syncthreads();

    // Write data from shared memory to global memory
    if (i < n) {
        a[i] = sharedData[tid];
    }
}