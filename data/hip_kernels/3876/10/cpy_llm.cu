#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cpy(int *a, int *b, int n) {
    // Calculate the global thread index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize shared memory for thread sums
    __shared__ int shared_sum[1024];
    shared_sum[threadIdx.x] = 0;

    // Perform operations in a loop with coalesced memory access pattern
    while (i < n) {
        shared_sum[threadIdx.x] += b[i];
        i += blockDim.x * gridDim.x;
    }
    
    // Synchronize threads within the block
    __syncthreads();
    
    // Reduce shared memory sums to a single sum
    if (threadIdx.x == 0) {
        int block_sum = 0;
        for (int j = 0; j < blockDim.x; j++) {
            block_sum += shared_sum[j];
        }
        atomicAdd(a, block_sum);
    }
}