#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce(int *a, int *b, int n) {
    // Calculate global index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check index boundary
    if (i < n) {
        // Perform atomic addition to avoid race conditions
        atomicAdd(b, a[i]);
    }
}