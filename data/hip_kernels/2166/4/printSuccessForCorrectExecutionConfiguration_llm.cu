#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void printSuccessForCorrectExecutionConfiguration()
{
    // Optimized to use block and thread index directly
    if (blockIdx.x == gridDim.x - 1 && threadIdx.x == blockDim.x - 1)
    {
        printf("Success!\n");
    }
}