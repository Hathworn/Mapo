#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy(int * a, int * b, int * c)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid-strided loop for better memory throughput
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < N; i += stride) {
        c[i] = 2 * a[i] + b[i]; // Perform SAXPY operation
    }
}