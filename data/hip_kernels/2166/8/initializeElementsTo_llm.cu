#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initializeElementsTo(int initialValue, int *a, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x; // Calculate grid stride
    
    for (; i < N; i += gridSize) // Use grid-stride loop for better utilization
    {
        a[i] = initialValue;
    }
}