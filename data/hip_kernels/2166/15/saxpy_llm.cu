#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy(int * a, int * b, int * c, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Fixed index calculation by using + operator
    if (tid < N) // Unchanged bounds check
    {
        c[tid] = 2 * a[tid] + b[tid]; // Simple calculation remained intact
    }
}