#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMulGPU(int *a, int *b, int *c, int N) {
    // Use shared memory to optimize memory access
    __shared__ int shared_a[TILE_DIM][TILE_DIM];
    __shared__ int shared_b[TILE_DIM][TILE_DIM];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int val = 0;

    for (int tile = 0; tile < (N + TILE_DIM - 1) / TILE_DIM; ++tile) {
        // Load data into shared memory
        if (row < N && tile * TILE_DIM + threadIdx.x < N)
            shared_a[threadIdx.y][threadIdx.x] = a[row * N + tile * TILE_DIM + threadIdx.x];
        else
            shared_a[threadIdx.y][threadIdx.x] = 0;

        if (col < N && tile * TILE_DIM + threadIdx.y < N)
            shared_b[threadIdx.y][threadIdx.x] = b[(tile * TILE_DIM + threadIdx.y) * N + col];
        else
            shared_b[threadIdx.y][threadIdx.x] = 0;

        __syncthreads();

        // Perform the multiplication
        for (int k = 0; k < TILE_DIM; ++k)
            val += shared_a[threadIdx.y][k] * shared_b[k][threadIdx.x];

        __syncthreads();
    }

    // Store the result back to global memory
    if (row < N && col < N)
        c[row * N + col] = val;
}