#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy(int * a, int * b, int * c, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop with stride to process all elements
    for (int i = tid; i < n; i += stride)
    {
        c[i] = 2 * a[i] + b[i];
    }
}