#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel with proper indexing
__global__ void firstParallel()
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if (idx == 0) // Ensure only one thread does the work
    {
        printf("This is running in parallel.\n");
    }
}