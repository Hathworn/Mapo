#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void loop()
{
    // Use of 'const' and 'unsigned int' as index won't change
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Minimize printf calls; Uncomment if printing is necessary
    // printf("%d\n", i);
}
```
