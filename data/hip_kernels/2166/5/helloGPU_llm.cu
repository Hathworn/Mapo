#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloGPU()
{
    // Use block and thread indices to reduce redundant prints
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        printf("Hello from the GPU.\n");
    }
}