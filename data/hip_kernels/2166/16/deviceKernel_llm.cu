#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deviceKernel(int *a, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Use conditional check to avoid out-of-bounds access
    if (idx < N) {
        a[idx] = 1;
    }
}