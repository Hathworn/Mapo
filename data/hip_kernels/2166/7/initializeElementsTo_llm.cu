#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initializeElementsTo(int initialValue, int *a, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) // Ensure threads do not access out-of-bounds memory
    {
        a[i] = initialValue;
    }
}