#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N  64

__global__ void matrixMulGPU(int * a, int * b, int * c)
{
    // Calculate row index of the element
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    // Calculate column index of the element
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize the sum variable
    int sum = 0;
    
    // Perform multiplication and accumulate
    for (int k = 0; k < N; k++) {
        sum += a[row * N + k] * b[k * N + col];
    }
    
    // Write the result to the output matrix
    c[row * N + col] = sum;
}