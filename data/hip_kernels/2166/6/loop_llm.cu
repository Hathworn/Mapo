#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use cooperative grid-stride loop to ensure full utilization of the GPU
__global__ void loop()
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global index
    
    for (int i = idx; i < N; i += blockDim.x * gridDim.x) { // Grid-stride loop for multiple iterations
        printf("This is iteration number %d\n", i);
    }
}