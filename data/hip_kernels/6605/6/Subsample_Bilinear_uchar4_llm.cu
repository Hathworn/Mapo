#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subsample_Bilinear_uchar4(hipTextureObject_t uchar4_tex, uchar4 *dst, int dst_width, int dst_height, int dst_pitch, int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;

        // Pre-calculate and optimize weight computations
        float inv_hscale = 1.0f / hscale;
        float inv_vscale = 1.0f / vscale;
        
        // Use these values directly for filtering
        float dx = inv_hscale * 0.5f;
        float dy = inv_vscale * 0.5f;

        uchar4 c0 = tex2D<uchar4>(uchar4_tex, xi - dx, yi - dy);
        uchar4 c1 = tex2D<uchar4>(uchar4_tex, xi + dx, yi - dy);
        uchar4 c2 = tex2D<uchar4>(uchar4_tex, xi - dx, yi + dy);
        uchar4 c3 = tex2D<uchar4>(uchar4_tex, xi + dx, yi + dy);

        int4 res;
        res.x = ((int)c0.x + (int)c1.x + (int)c2.x + (int)c3.x + 2) >> 2;
        res.y = ((int)c0.y + (int)c1.y + (int)c2.y + (int)c3.y + 2) >> 2;
        res.z = ((int)c0.z + (int)c1.z + (int)c2.z + (int)c3.z + 2) >> 2;
        res.w = ((int)c0.w + (int)c1.w + (int)c2.w + (int)c3.w + 2) >> 2;
        
        dst[yo * dst_pitch + xo] = make_uchar4((unsigned char)res.x, (unsigned char)res.y, (unsigned char)res.z, (unsigned char)res.w);
    }
}