#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Thumbnail_ushort2(hipTextureObject_t ushort2_tex, int *histogram, int src_width, int src_height)
{
    // Use shared memory for histogram accumulation within block
    __shared__ int localHist[512];
    int localIndex = threadIdx.y * blockDim.x + threadIdx.x;
    if (localIndex < 512) localHist[localIndex] = 0;
    __syncthreads();

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < src_height && x < src_width)
    {
        ushort2 pixel = tex2D<ushort2>(ushort2_tex, x, y);
        atomicAdd(&localHist[(pixel.x + 128) >> 8], 1);
        atomicAdd(&localHist[256 + (pixel.y + 128) >> 8], 1);
    }
    __syncthreads();

    // Aggregate local histogram to global memory
    if (localIndex < 512)
    {
        atomicAdd(&histogram[localIndex], localHist[localIndex]);
    }
}