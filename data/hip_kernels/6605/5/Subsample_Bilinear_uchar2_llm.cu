#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subsample_Bilinear_uchar2(hipTextureObject_t uchar2_tex, uchar2 *dst, int dst_width, int dst_height, int dst_pitch2, int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        // Precompute and reuse scale factors for efficiency
        float hscale = __fdividef(src_width, dst_width);
        float vscale = __fdividef(src_height, dst_height);
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;
        
        // Simplifying min and max using clamp
        float wh = clamp(0.5f * (hscale - 1.0f), 0.0f, 1.0f);
        float wv = clamp(0.5f * (vscale - 1.0f), 0.0f, 1.0f);

        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);

        // Fetch all texture data with reduced function calls
        uchar2 c0 = tex2D<uchar2>(uchar2_tex, xi - dx, yi - dy);
        uchar2 c1 = tex2D<uchar2>(uchar2_tex, xi + dx, yi - dy);
        uchar2 c2 = tex2D<uchar2>(uchar2_tex, xi - dx, yi + dy);
        uchar2 c3 = tex2D<uchar2>(uchar2_tex, xi + dx, yi + dy);

        // Optimizing computation by simplifying averaging
        int2 uv;
        uv.x = (__int2(c0.x) + c1.x + c2.x + c3.x + 2) >> 2;
        uv.y = (__int2(c0.y) + c1.y + c2.y + c3.y + 2) >> 2;

        // Output the result to destination array
        dst[yo * dst_pitch2 + xo] = make_uchar2((unsigned char)uv.x, (unsigned char)uv.y);
    }
}