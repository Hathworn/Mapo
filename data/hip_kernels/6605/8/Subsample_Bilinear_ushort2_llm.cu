#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subsample_Bilinear_ushort2(hipTextureObject_t ushort2_tex, ushort2 *dst, int dst_width, int dst_height, int dst_pitch2, int src_width, int src_height)
{
    // Compute normalized texture coordinates
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = __fdividef(src_width, dst_width);
        float vscale = __fdividef(src_height, dst_height);
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;

        // Precompute weights based on scale factors
        float wh = __saturatef(0.5f * (hscale - 1.0f));
        float wv = __saturatef(0.5f * (vscale - 1.0f));
        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);

        // Fetch four surrounding texels from texture
        ushort2 c0 = tex2D<ushort2>(ushort2_tex, xi-dx, yi-dy);
        ushort2 c1 = tex2D<ushort2>(ushort2_tex, xi+dx, yi-dy);
        ushort2 c2 = tex2D<ushort2>(ushort2_tex, xi-dx, yi+dy);
        ushort2 c3 = tex2D<ushort2>(ushort2_tex, xi+dx, yi+dy);

        // Average the texels for bilinear subsampling
        int2 uv;
        uv.x = ((int)c0.x + (int)c1.x + (int)c2.x + (int)c3.x + 2) >> 2;
        uv.y = ((int)c0.y + (int)c1.y + (int)c2.y + (int)c3.y + 2) >> 2;
        
        // Store the resulting pixel in the destination buffer
        dst[yo * dst_pitch2 + xo] = make_ushort2((unsigned short)uv.x, (unsigned short)uv.y);
    }
}