#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Thumbnail_uchar(hipTextureObject_t uchar_tex, int *histogram, int src_width, int src_height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Use shared memory to reduce global memory access; assume histogram size is small enough
    __shared__ int local_histogram[256];
    if (threadIdx.x < 256)  // Initialize shared memory histogram
    {
        local_histogram[threadIdx.x] = 0;
    }
    __syncthreads();

    if (y < src_height && x < src_width)
    {
        unsigned char pixel = tex2D<unsigned char>(uchar_tex, x, y);
        atomicAdd(&local_histogram[pixel], 1);
    }

    __syncthreads();

    if (threadIdx.x < 256)  // Update global memory histogram
    {
        atomicAdd(&histogram[threadIdx.x], local_histogram[threadIdx.x]);
    }
}