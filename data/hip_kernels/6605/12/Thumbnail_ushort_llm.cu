#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Thumbnail_ushort(hipTextureObject_t ushort_tex, int *histogram, int src_width, int src_height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check boundary conditions
    if (y < src_height && x < src_width)
    {
        // Improved texture fetching and computation with a temporary variable
        unsigned short pixel_val = tex2D<unsigned short>(ushort_tex, x, y);
        unsigned short pixel = (pixel_val + 128) >> 8;

        // Use atomic addition for histogram update
        atomicAdd(&histogram[pixel], 1);
    }
}