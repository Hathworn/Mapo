#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subsample_Bilinear_uchar(hipTextureObject_t uchar_tex, unsigned char *dst, int dst_width, int dst_height, int dst_pitch, int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = static_cast<float>(src_width) / dst_width;
        float vscale = static_cast<float>(src_height) / dst_height;
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;

        // Pre-compute reused values for texture fetching
        float dx = 0.5f * fmaxf(0.0f, fminf(hscale - 1.0f, 1.0f)) / (0.5f + 0.5f * fmaxf(0.0f, fminf(hscale - 1.0f, 1.0f)));
        float dy = 0.5f * fmaxf(0.0f, fminf(vscale - 1.0f, 1.0f)) / (0.5f + 0.5f * fmaxf(0.0f, fminf(vscale - 1.0f, 1.0f)));

        // Use batched texture fetches
        unsigned char y0 = tex2D<unsigned char>(uchar_tex, xi - dx, yi - dy);
        unsigned char y1 = tex2D<unsigned char>(uchar_tex, xi + dx, yi - dy);
        unsigned char y2 = tex2D<unsigned char>(uchar_tex, xi - dx, yi + dy);
        unsigned char y3 = tex2D<unsigned char>(uchar_tex, xi + dx, yi + dy);

        // Use more efficient averaging
        dst[yo * dst_pitch + xo] = static_cast<unsigned char>((y0 + y1 + y2 + y3 + 2) >> 2);
    }
}