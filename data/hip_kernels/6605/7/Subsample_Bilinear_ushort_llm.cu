#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subsample_Bilinear_ushort(hipTextureObject_t ushort_tex, unsigned short *dst, int dst_width, int dst_height, int dst_pitch, int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = __fdividef(src_width, dst_width); // Use fast division
        float vscale = __fdividef(src_height, dst_height); // Use fast division
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;

        // Calculate weight factors, optimized with lower bounds
        float wh = fmaxf(0.0f, fminf(0.5f * (hscale - 1.0f), 1.0f));
        float wv = fmaxf(0.0f, fminf(0.5f * (vscale - 1.0f), 1.0f));

        // Directly precompute bilinear weights
        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);

        // Minimize texture fetch operations; use shared memory if possible
        int y0 = tex2D<unsigned short>(ushort_tex, xi - dx, yi - dy);
        int y1 = tex2D<unsigned short>(ushort_tex, xi + dx, yi - dy);
        int y2 = tex2D<unsigned short>(ushort_tex, xi - dx, yi + dy);
        int y3 = tex2D<unsigned short>(ushort_tex, xi + dx, yi + dy);

        // Store result using efficient bit shifting
        dst[yo * dst_pitch + xo] = (unsigned short)((y0 + y1 + y2 + y3 + 2) >> 2);
    }
}