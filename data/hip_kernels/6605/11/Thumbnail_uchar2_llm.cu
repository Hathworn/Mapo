#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Thumbnail_uchar2(hipTextureObject_t uchar2_tex, int *histogram, int src_width, int src_height)
{
    // Use shared memory to reduce global memory accesses
    __shared__ int local_histogram[512];
    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    
    // Initialize shared memory
    if (tid < 512)
    {
        local_histogram[tid] = 0;
    }
    __syncthreads();
    
    if (y < src_height && x < src_width)
    {
        uchar2 pixel = tex2D<uchar2>(uchar2_tex, x, y);
        atomicAdd(&local_histogram[pixel.x], 1);
        atomicAdd(&local_histogram[256 + pixel.y], 1);
    }
    __syncthreads();
    
    // Write results back to global memory
    if (tid < 512)
    {
        atomicAdd(&histogram[tid], local_histogram[tid]);
    }
}