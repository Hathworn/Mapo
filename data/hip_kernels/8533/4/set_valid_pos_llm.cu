#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_valid_pos(int32_t* pos_buff, int32_t* count_buff, const int32_t entry_count) {
    const int32_t start = threadIdx.x + blockDim.x * blockIdx.x;
    const int32_t step = blockDim.x * gridDim.x;

    // Use shared memory for better performance
    __shared__ int32_t shared_count[256]; // Assume a typical blockDim.x size

    for (int32_t i = start; i < entry_count; i += step) {
        if (VALID_POS_FLAG == pos_buff[i]) {
            // Load previous count value into shared memory to minimize memory access time
            shared_count[threadIdx.x] = !i ? 0 : count_buff[i - 1];
            syncthreads(); // Ensure all threads have loaded their value before using
            pos_buff[i] = shared_count[threadIdx.x];
        }
    }
}