#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void columnarize_groups(int8_t* columnar_buffer, const int8_t* rowwise_buffer, const size_t row_count, const size_t col_count, const size_t* col_widths, const size_t row_size) {
    const auto thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the valid thread index is within bounds
    if (thread_index >= row_count) return;

    auto read_ptr = rowwise_buffer + thread_index * row_size;
    auto col_base = columnar_buffer;

    for (size_t i = 0; i < col_count; ++i) {
        switch (col_widths[i]) {
            case 8: {
                // Use efficient pointer arithmetic for 64-bit data
                reinterpret_cast<int64_t*>(col_base)[thread_index] = *reinterpret_cast<const int64_t*>(read_ptr);
                break;
            }
            case 4: {
                // Use efficient pointer arithmetic for 32-bit data
                reinterpret_cast<int32_t*>(col_base)[thread_index] = *reinterpret_cast<const int32_t*>(read_ptr);
                break;
            }
            default:
                break;
        }
        col_base += col_widths[i] * row_count;
        read_ptr += col_widths[i];
    }
}