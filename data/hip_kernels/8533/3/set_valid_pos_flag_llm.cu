#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_valid_pos_flag(int32_t* pos_buff, const int32_t* count_buff, const int32_t entry_count) {
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate the global thread index
    int32_t step = blockDim.x * gridDim.x;  // Calculate step size

    for (int32_t i = idx; i < entry_count; i += step) {
        if (count_buff[i]) {
            pos_buff[i] = VALID_POS_FLAG;  // Set position flag if count is valid
        }
    }
}