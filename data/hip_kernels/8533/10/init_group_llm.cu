#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_group(int8_t* groups, const size_t group_count, const size_t col_count, const size_t* col_widths, const size_t* init_vals) {
    const auto thread_index = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;
    if (thread_index >= group_count) {
        return;
    }
    
    // Loop unrolling for improved performance
    int8_t* group_ptr = groups + thread_index;
    for (size_t i = 0; i < col_count; ++i) {
        size_t width = col_widths[i];
        if (width == 4) {
            *reinterpret_cast<uint32_t*>(group_ptr + i * group_count * width) = *reinterpret_cast<const uint32_t*>(init_vals + i);
        } else if (width == 8) {
            *reinterpret_cast<size_t*>(group_ptr + i * group_count * width) = init_vals[i];
        }
    }
}