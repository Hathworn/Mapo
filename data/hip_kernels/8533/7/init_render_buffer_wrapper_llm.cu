#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_render_buffer(int64_t* render_buffer, const uint32_t qw_count) {
    const uint32_t start = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t step = blockDim.x * gridDim.x;

    // Each thread processes multiple elements to improve parallel efficiency
    for (uint32_t i = start; i < qw_count; i += step) {
        render_buffer[i] = EMPTY_KEY_64;
    }
}