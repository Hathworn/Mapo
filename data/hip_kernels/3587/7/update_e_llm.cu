#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void update_e(int objs, double* e, double* kval, double b_old, double b_new, int i, int j, int yi, int yj, double ai_old, double ai_new, double aj_old, double aj_new) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < objs) {
        // Pre-calculate deltas for efficiency
        double delta_b = b_new - b_old;
        double delta_ai = ai_new - ai_old;
        double delta_aj = aj_new - aj_old;

        // Pre-fetch repetitive kval indices
        double kval_i = kval[i * objs + id];
        double kval_j = kval[j * objs + id];

        // Update computation using pre-calculated values
        double val = e[id];
        val += delta_b;
        val += yi * kval_i * delta_ai;
        val += yj * kval_j * delta_aj;
        e[id] = val;
    }
}