#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void calc_linear_kernel(int objs, int coords, double* x, double* out) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int i = id / objs;
    int j = id % objs;

    if (i < objs) {
        double r = 0.0;

        // Use shared memory to reduce global memory access latency
        extern __shared__ double shared_x[];
        for (int k = threadIdx.x; k < coords * objs; k += blockDim.x) {
            shared_x[k] = x[k];
        }
        __syncthreads();

        for (int k = 0; k < coords; k++) {
            r += shared_x[objs * k + i] * shared_x[objs * k + j];
        }

        out[objs * i + j] = r;
    }
}