#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void calc_linear_kernel(int objs, int coords, double* x, double* out) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int total_threads = blockDim.x * gridDim.x;
    
    // Use stride to distribute work among threads efficiently
    for (int id = tid; id < objs * objs; id += total_threads) {
        int i = id / objs;
        int j = id % objs;
        
        if (i < objs) {
            double r = 0.0;
            for (int k = 0; k < coords; ++k) {
                r += x[objs * k + i] * x[objs * k + j];
            }
            out[objs * i + j] = r;
        }
    }
}