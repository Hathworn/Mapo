#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void calc_e(int objs, double* a, double b, int* y, double* kval, double* e) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < objs) {
        double fx = b;
        for (int i = 0; i < objs; i++) {
            // Use shared memory to improve memory access pattern
            __shared__ double shared_a_y[/* appropriate size based on usage */];
            if (threadIdx.x == 0) {
                shared_a_y[i] = a[i] * y[i];
            }
            __syncthreads();
            
            fx += shared_a_y[i] * kval[i * objs + id];
        }
        e[id] = fx - y[id];
    }
}