#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void calc_e(int objs, double* a, double b, int* y, double* kval, double* e) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < objs) {
        double fx = b;
        // Use shared memory for coalesced access
        __shared__ double shared_a[256];
        __shared__ int shared_y[256];

        for (int i = threadIdx.x; i < objs; i += blockDim.x) {
            if (threadIdx.x < objs) {
                shared_a[threadIdx.x] = a[threadIdx.x];
                shared_y[threadIdx.x] = y[threadIdx.x];
            }
            __syncthreads();

            for (int offset = 0; offset < objs; offset += blockDim.x) {
                int idx = offset + threadIdx.x;
                if (idx < objs) {
                    fx += shared_a[threadIdx.x] * shared_y[threadIdx.x] * kval[idx * objs + id];
                }
                __syncthreads();
            }
        }
        e[id] = fx - y[id];
    }
}