#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void calc_linear_kernel_predict(int objs, int coords, double* x, int objs_train, double* x_train, double* out) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int i = idx / objs;
    int j = idx % objs;
    if (i < objs_train) {
        double r = 1.0;
        int offset_i = coords * i; // Precompute offset for x_train
        int offset_j = coords * j; // Precompute offset for x

        // Unroll loop to increase performance
        for (int k = 0; k < coords; k += 2) {
            r += x_train[offset_i + k] * x[offset_j + k];
            if (k + 1 < coords) {  // Handle odd number of coords
                r += x_train[offset_i + k + 1] * x[offset_j + k + 1];
            }
        }
        out[idx] = r;
    }
}