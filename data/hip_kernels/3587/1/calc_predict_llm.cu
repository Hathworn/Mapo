#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void calc_predict(int objs, int objs_train, double* __restrict__ a, double b, int* __restrict__ y_train, double* __restrict__ kval, int* __restrict__ y) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < objs) {
        double fx = b;
        for (int i = 0; i < objs_train; i++) {
            // Coalesced memory access for kval
            fx += a[i] * y_train[i] * kval[id * objs_train + i]; // Changed indexing for kval
        }
        y[id] = fx >= 0 ? 1 : -1;
    }
}