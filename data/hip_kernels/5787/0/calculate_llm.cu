#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 512

__global__ void calculate(int *a, int *b, int *c){
    int idx = threadIdx.x; // Cache thread index for reuse
    int temp = a[idx] + 2; // Precompute repeated expression
    c[idx] = (temp + b[idx]) * 3; // Use cached value in final computation
}