#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

static __device__ float E = 2.718281828;

__global__ void transformBboxSQDKernel(float *delta, float *anchor, float *res, int block_size)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int di = threadId * 4; // Calculate index for 4-element accesses

    if (di < block_size * 4) { // Ensure index is within bounds
        float d[4] = {delta[di], delta[di+1], delta[di+2], delta[di+3]};
        float a[4] = {anchor[di], anchor[di+1], anchor[di+2], anchor[di+3]};
        
        // Unroll calculations for better optimization
        float cx = a[0] + d[0] * a[2];
        float cy = a[1] + d[1] * a[3];
        float w = a[2] * (d[2] < 1.0f ? __expf(d[2]) : d[2] * E);
        float h = a[3] * (d[3] < 1.0f ? __expf(d[3]) : d[3] * E);
        
        // Store results in global memory
        res[di] = cx - w * 0.5f;
        res[di+1] = cy - h * 0.5f;
        res[di+2] = cx + w * 0.5f;
        res[di+3] = cy + h * 0.5f;
    }
}