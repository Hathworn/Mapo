#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

static __device__ float E = 2.718281828;

__global__ void multiplyElementKernel(float *src1, float *src2, float *dst, int block_size)
{
    // Calculate the global index
    int di = blockIdx.x * block_size + threadIdx.x;
    
    // Check bounds within global size
    if (di < block_size * gridDim.x) {
        // Perform element-wise multiplication
        dst[di] = src1[di] * src2[di];
    }
}