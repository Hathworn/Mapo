#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

static __device__ float E = 2.718281828;

__global__ void reduceArgMaxKernel(float *src, float *dst, float *arg, int dim_size, int block_size)
{
    // Each thread processes its own block index
    int di = blockIdx.x * block_size + threadIdx.x;
    int si = di * dim_size;
    
    // Initialize local memory for reduction
    float max = src[si];
    int maxi = 0;

    // Efficient iteration over dimension size
    for (int i = 1; i < dim_size; i++) {
        float now = src[si + i];
        if (now > max) {
            max = now;
            maxi = i;
        }
    }
    
    // Save results in global memory
    dst[di] = max;
    arg[di] = maxi;
}