#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elevar_al_cuadrado(float * d_salida, float * d_entrada) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load input value
    float f = d_entrada[idx];
    
    // Compute square
    d_salida[idx] = f * f;
}
```
