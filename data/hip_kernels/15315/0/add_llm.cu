#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define N 100

__global__ void add(int *a, int *c)
{
    // Use thread index to improve parallelism
    int tID = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tID < N)
    {
        // Direct computation without additional variable
        c[tID] = 3 * a[tID];
    }
}