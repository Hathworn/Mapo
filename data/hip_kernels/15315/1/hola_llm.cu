#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hola() {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate unique thread index
    if (idx == 0) {  // Ensure only one thread executes the print statement
        printf("Hola mundo! Soy un thread en el bloque %d\n", blockIdx.x);
    }
}