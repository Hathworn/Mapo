#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prefSumBinTreeCudaMulti(float *a, int n) {
    __shared__ float shm[CUDA_THREAD_NUM];
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int offset = CUDA_THREAD_NUM * bid;
    
    // Load data into shared memory
    shm[tid] = a[offset + tid];
    __syncthreads();
    
    // Binary tree reduction
    for (int stride = 1; stride <= n; stride *= 2) {
        if ((tid + 1) % (2 * stride) == 0) {
            shm[tid] += shm[tid - stride];
        }
        __syncthreads();
    }
    
    // Down-sweep phase
    for (int stride = n / 2; stride > 0; stride /= 2) {
        if ((tid + 1) % (2 * stride) == 0) {
            shm[tid] = shm[tid] + shm[tid - stride];
        }
        __syncthreads();
    }
    
    // Write results back to global memory
    a[offset + tid] = shm[tid];
}