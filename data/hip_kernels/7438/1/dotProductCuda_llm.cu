#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;
#define CUDA_THREAD_NUM 1024
// must be a multiple of 2

void dotProductCPU();
__global__ void dotProductCuda(float *a, float *b, float *c) {
    __shared__ float se[CUDA_THREAD_NUM];

    // Calculate a.*b using the entire warp
    int tid = threadIdx.x + blockIdx.x * CUDA_THREAD_NUM;
    se[threadIdx.x] = a[tid] * b[tid];
    __syncthreads();

    // Improved Sum Reduction using warp shuffles
    for (unsigned int stride = warpSize / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            se[threadIdx.x] += se[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write block result to c
    if (threadIdx.x == 0) {
        c[blockIdx.x] = se[0];
    }
}