#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculateResidual_CUDA(float *a, float *b, float *c) {
    extern __shared__ float se[]; // Use dynamic shared memory

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int n = blockDim.x;
    int index = tid + bid * n;

    // Calculate the absolute difference and store in shared memory
    se[tid] = fabsf(a[index] - b[index]);
    __syncthreads();

    // Reduction using binary tree algorithm
    for (int offset = n / 2; offset > 0; offset >>= 1) {
        if (tid < offset) {
            se[tid] += se[tid + offset];
        }
        __syncthreads();
    }

    // Atomic addition of the block's result to global memory
    if (tid == 0) {
        atomicAdd(c, se[0]);
    }
}