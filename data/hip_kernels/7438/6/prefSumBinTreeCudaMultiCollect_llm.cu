#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prefSumBinTreeCudaMultiCollect(float *a, int n, int numBlocks) {
    __shared__ float shm[CUDA_THREAD_NUM];
    int tid = threadIdx.x;
    
    // Load data into shared memory
    shm[tid] = a[tid];
    __syncthreads();

    // Perform upsweep phase of the parallel prefix sum
    for (int stride = 1; stride <= n / 2; stride *= 2) {
        int index = (tid + 1) * stride * 2 - 1;
        if (index < n) {
            shm[index] += shm[index - stride];
        }
        __syncthreads();
    }

    // Clear the last element for downsweep phase
    if (tid == 0) {
        shm[n - 1] = 0;
    }
    __syncthreads();

    // Perform downsweep phase
    for (int stride = n / 2; stride > 0; stride /= 2) {
        int index = (tid + 1) * stride * 2 - 1;
        if (index < n) {
            float temp = shm[index - stride];
            shm[index - stride] = shm[index];
            shm[index] += temp;
        }
        __syncthreads();
    }

    // Write results back to global memory
    a[tid] = shm[tid];
}