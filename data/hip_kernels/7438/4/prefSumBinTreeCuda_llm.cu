#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prefSumBinTreeCuda(float *a, int n) {
    __shared__ float shm[CUDA_THREAD_NUM];
    int tid = threadIdx.x;
    int dot = 2; // Initial depth of tree

    // Load data into shared memory with stride of dot
    if ((tid + 1) % dot == 0) {
        shm[tid] = a[tid] + a[tid - 1];
    }
    dot *= 2;
    __syncthreads();

    // Compute prefix sum using a binary tree approach
    while (dot <= n) {
        if ((tid + 1) % dot == 0) {
            shm[tid] = shm[tid] + shm[tid - dot / 2];
        }
        dot *= 2;
        __syncthreads();
    }

    // Propagate back results to shared memory
    dot /= 2;
    while (dot > 2) {
        if ((tid + 1) % dot == 0) {
            if ((tid + 1) / dot != 1) {
                shm[tid - dot / 2] += shm[tid - dot];
            }
        }
        dot /= 2;
        __syncthreads();
    }

    // Write results back to array a
    if ((tid + 1) % 2 == 0) {
        a[tid] = shm[tid];
    } else if (tid > 0) {
        a[tid] += shm[tid - 1];
    }
}