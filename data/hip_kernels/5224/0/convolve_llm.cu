#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

texture<float, hipTextureType2D> tex_img;
texture<float, hipTextureType2D> tex_kernel;

__global__ void convolve(const int input_width, const int input_height, const int kernel_radius, float *output) {
    
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if ((col < input_height) && (row < input_width)) {
        float value = 0.0f;

        // Unroll loop to reduce overhead and improve parallel execution
        #pragma unroll
        for (int y = -kernel_radius; y <= kernel_radius; ++y) {
            #pragma unroll
            for (int x = -kernel_radius; x <= kernel_radius; ++x) {
                // Use texture fetches, leverage caching mechanism
                value += tex2D(tex_img, row + x, col + y) * tex2D(tex_kernel, x + kernel_radius, y + kernel_radius);
            }
        }

        int idx = col * input_width + row;
        output[idx] = value; // Coalesced write
    }
}