#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void array_copy(float* a, float* c, int mx, int my, int mz, int sx, int sy, int sz, int ox, int oy, int oz) {
    int pnum = blockIdx.x * blockDim.x + threadIdx.x;  // thread index

    // Pre-calculate reusable constants
    int half_sx = sx / 2;
    int half_sy = sy / 2;

    int px = pnum % half_sx;  // x position of the complex number
    int py = pnum / half_sx;  // y position of the complex number

    // Exit if out of bounds
    if (px >= sx || py >= half_sy) return;

    int ids = 2 * (px + py * sx);  // source index
    int idd = 2 * ((ox + px) + (oy + py) * sx);  // destination index

    // Efficient in-register swap of values
    float tmpR = c[idd];
    float tmpI = c[idd + 1];

    c[idd] = a[ids];
    c[idd + 1] = a[ids + 1];

    a[ids] = tmpR;
    a[ids + 1] = tmpI;
}