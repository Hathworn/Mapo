#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rotate2(float *a, float b, float *c, int sx, int sy, int sz, int dx, int dy, int dz)
{
    int ids = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique thread ID

    if (ids >= sx * sy * sz) return; // Boundary check to prevent out-of-bounds access

    // Compute wrapped coordinates with offsets
    int x = (ids + dx) % sx;
    int y = ((ids / sx) + dy) % sy;
    int z = ((ids / (sx * sy)) + dz) % sz;

    // Compute destination index
    int idd = x + sx * y + sx * sy * z;

    // Perform rotation and scaling operation
    c[idd] = b * a[ids];
}