#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_carr(float br, float bi, float * c, int N)
{
    // Calculate unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (idx < N) {
        int idc = idx * 2;  // Calculate position in c

        // Set values in c array
        c[idc] = br;
        c[idc + 1] = bi;
    }
}