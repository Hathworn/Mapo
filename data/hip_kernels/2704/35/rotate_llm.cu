#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// To suppress the unused variable argument for ARM targets
#pragma diag_suppress 177

#ifndef NAN
#define NAN (0.0/0.0)
#endif

#define ACCU_ARRTYPE double
#define IMUL(a, b) __mul24(a, b)

#ifndef CUIMAGE_REDUCE_THREADS
#define CUIMAGE_REDUCE_THREADS 512
#endif

#define NBLOCKS(N,blockSize) (N/blockSize+(N%blockSize==0?0:1))
#define NBLOCKSL(N,blockSize) 1

__global__ void rotate(float* a, float b, float* c, size_t sx, size_t sy, size_t sz, size_t dx, size_t dy, size_t dz, size_t ux, size_t uy, size_t uz) {
    size_t id = ((blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x);
    size_t Processes = blockDim.x * gridDim.x;
    size_t chains = ux * uy * uz;
    size_t N = sx * sy * sz;
    size_t length = N / chains;
    size_t steps = (N + Processes - 1) / Processes; // Ensure all data is covered

    float swp, nswp;

    // Optimize the computation process using loops and conditions
    for (size_t idx = id * steps; idx < min(N, (id + 1) * steps); ++idx) {
        size_t step = idx;
        size_t nl = step % length;
        size_t nx = (step / length) % ux;
        size_t ny = (step / (length * ux)) % uy;
        size_t nz = (step / (length * ux * uy)) % uz;

        size_t x = (nx + nl * dx) % sx;
        size_t y = (ny + nl * dy) % sy;
        size_t z = (nz + nl * dz) % sz;
        size_t idd = x + sx * y + sx * sy * z;

        if (nl < length - 1) {
            swp = a[idd];
            __syncthreads();
            x = (x + dx) % sx;
            y = (y + dy) % sy;
            z = (z + dz) % sz;
            idd = x + sx * y + sx * sy * z;
            nswp = a[idd];
            __syncthreads();
            c[idd] = swp + 0.1;
            swp = nswp;
        }

        // Final write to avoid race conditions
        c[idd] = swp + 0.1;
    }
}