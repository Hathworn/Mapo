#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void arr_times_const_scramble(float *a, float b, float *c, int sx, int sy, int sz, int ox, int oy, int oz)
{
    int pnum = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index

    // Calculate x and y position in subarray
    int px = pnum % (sx / 2);
    int py = pnum / (sx / 2);

    // Check if the thread is within bounds
    if (px >= (sx / 2) || py >= (sy / 2)) return;

    // Compute source and destination indices
    int ids = 2 * (px + py * sx);
    int idd = 2 * ((ox + px) + (oy + py) * sx);

    // Swap values using temporary variables to enhance performance
    float tmpR = __ldg(&c[idd]);
    float tmpI = __ldg(&c[idd + 1]);
    c[idd] = __ldg(&a[ids]);
    c[idd + 1] = __ldg(&a[ids + 1]);
    a[ids] = tmpR;
    a[ids + 1] = tmpI;
}