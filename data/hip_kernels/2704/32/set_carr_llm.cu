#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// To suppress the unused variable argument for ARM targets
#pragma diag_suppress 177

#ifndef NAN   // should be part of math.h
#define NAN (0.0/0.0)
#endif

#define ACCU_ARRTYPE double  // Type of the tempory arrays for reduce operations
#define IMUL(a, b) __mul24(a, b)

#ifndef CUIMAGE_REDUCE_THREADS
#define CUIMAGE_REDUCE_THREADS 512
#endif

#define NBLOCKS(N,blockSize) (N/blockSize+(N%blockSize==0?0:1))

#define NBLOCKSL(N,blockSize) 1

__global__ void set_carr(float br, float bi, float * c, size_t N)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Optimize boundary check
    if (idx < N)
    {
        size_t idc = idx * 2;
        c[idc] = br;
        c[idc + 1] = bi;
    }
}