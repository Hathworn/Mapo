#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// To suppress the unused variable argument for ARM targets
#pragma diag_suppress 177

#ifndef NAN   // should be part of math.h
#define NAN (0.0/0.0)
#endif

#define ACCU_ARRTYPE double  // Type of the tempory arrays for reduce operations
#define IMUL(a, b) __mul24(a, b)

#ifndef CUIMAGE_REDUCE_THREADS  // this can be defined at compile time via the flag NVCCFLAG='-D CUIMAGE_REDUCE_THREADS=512'
#define CUIMAGE_REDUCE_THREADS 512
#endif

#define NBLOCKS(N,blockSize) (N/blockSize+(N%blockSize==0?0:1))
#define NBLOCKSL(N,blockSize) 1

__global__ void set_arr(float b, float * c, size_t N)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplified calculation of linear thread index
    if (idx < N) { // Reduce the branch divergence by minimizing operations inside the conditional
        c[idx] = b; // Directly assign the value to array position
    }
}