#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void arr_times_const_checkerboard(float *a, float b, float *c, int N, int sx, int sy, int sz)
{
    // Calculate global thread index for flattening
    int ids = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread index is within bounds
    if (ids >= N) return;

    // Use integer division and modulus only once for px and py calculation
    int py = ids / (2 * sx);
    int px = (ids / 2) - (py * sx);

    // Calculate checkerboard pattern value
    float minus1 = 1 - 2 * ((px + py) % 2);

    // Compute scaling and store in output
    c[ids] = a[ids] * b * minus1;
}