#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#pragma diag_suppress 177

#ifndef NAN
#define NAN (0.0/0.0)
#endif

#define ACCU_ARRTYPE double
#define IMUL(a, b) __mul24(a, b)

#ifndef CUIMAGE_REDUCE_THREADS
#define CUIMAGE_REDUCE_THREADS 512
#endif

#define NBLOCKS(N,blockSize) (N/blockSize+(N%blockSize==0?0:1))
#define NBLOCKSL(N,blockSize) 1

__global__ void arr_times_const_checkerboard(float* a, float b, float* c, size_t N, size_t sx, size_t sy, size_t sz) {
    size_t ids = blockIdx.x * blockDim.x + threadIdx.x; // Flattened thread index
    if (ids >= N) return;  // Boundary check

    // Efficiently calculate px and py using integer division
    size_t px = (ids / 2) % sx;
    size_t py = (ids / (2 * sx)) % sy; // Correcting the calculation of py
    float minus1 = 1 - 2 * ((px + py) % 2);
    c[ids] = a[ids] * b * minus1;
}