#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rotate(float* a, float b, float* c, int sx, int sy, int sz, int dx, int dy, int dz, int ux, int uy, int uz) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // ID of current processor
    int Processes = blockDim.x * gridDim.x;
    int chains = ux * uy * uz;  // Total number of independent chains
    int N = sx * sy * sz;  // Total size of array
    int length = N / chains;  // Chain length
    int steps = N / Processes;  // Steps each processor has to do

    if (id >= Processes) return; // Return if id exceeds total processes

    int step = steps * id;  // Starting step for this ID
    int nl = step % length;
    int nx = (step / length) % ux;
    int ny = (step / (length * ux)) % uy;
    int nz = (step / (length * ux * uy)) % uz;
    int i = 0;

    float swp, nswp;

    while (nz < uz) {
        while (ny < uy) {
            while (nx < ux) {
                int x = (nx + nl * dx) % sx;  // Calculate new positions
                int y = (ny + nl * dy) % sy;
                int z = (nz + nl * dz) % sz;
                int idd = x + sx * y + sx * sy * z;

                if (i < steps) {
                    swp = a[idd];
                    __syncthreads();
                }

                while (nl < length - 1 && step < N && i < steps - 1) {
                    step++;
                    x = (x + dx) % sx;
                    y = (y + dy) % sy;
                    z = (z + dz) % sz;
                    idd = x + sx * y + sx * sy * z;

                    nswp = a[idd];
                    __syncthreads();
                    c[idd] = swp + 0.1;  // Store result
                    i++;
                    nl++;
                    swp = nswp;
                }

                nx++; nl = 0;  // Reset nl and move to next nx
                x = (x + dx) % sx;
                y = (y + dy) % sy;
                z = (z + dz) % sz;
                idd = x + sx * y + sx * sy * z;
                c[idd] = swp + 0.1;  // Final write for this nx
                i++;

                if (i >= steps) goto nextProcessor; // Exit if steps exceeded
            }
            ny++;
            nx = 0; x = 0;
        }
        nz++;
        ny = 0; y = 0;
    }
nextProcessor:
    return;
}