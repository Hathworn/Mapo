#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// To suppress the unused variable argument for ARM targets
#pragma diag_suppress 177

#ifndef NAN   // should be part of math.h
#define NAN (0.0/0.0)
#endif

#define ACCU_ARRTYPE double  // Type of the temporary arrays for reduce operations
#define IMUL(a, b) __mul24(a, b)

#ifndef CUIMAGE_REDUCE_THREADS
#define CUIMAGE_REDUCE_THREADS 512
#endif

#define NBLOCKS(N, blockSize) (N/blockSize+(N%blockSize==0?0:1))
#define NBLOCKSL(N, blockSize) 1

__global__ void arr_times_const_scramble(float* a, float b, float* c, size_t sx, size_t sy, size_t sz, size_t ox, size_t oy, size_t oz)
{
    // Calculate the global position for each thread
    size_t pnum = (blockDim.x * blockIdx.x + threadIdx.x) + (blockDim.x * gridDim.x * blockIdx.y);
    
    // Calculate positions in the array
    size_t px = pnum % (sx / 2);
    size_t py = pnum / (sx / 2);

    // Check if within bounds
    if (px >= (sx / 2) || py >= (sy / 2)) return;

    // Calculate indices for source and destination arrays
    size_t ids = 2 * (px + py * sx);
    size_t idd = 2 * ((ox + px) + (oy + py) * sx);

    // Perform element swap using temporary variables
    float tmpR = c[idd];
    float tmpI = c[idd + 1];
    c[idd] = a[ids];
    c[idd + 1] = a[ids + 1];
    a[ids] = tmpR;
    a[ids + 1] = tmpI;
}