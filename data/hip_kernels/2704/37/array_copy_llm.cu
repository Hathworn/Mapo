#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// To suppress the unused variable argument for ARM targets
#pragma diag_suppress 177

#ifndef NAN
#define NAN (0.0/0.0)
#endif

#define ACCU_ARRTYPE double
#define IMUL(a, b) __mul24(a, b)

#ifndef CUIMAGE_REDUCE_THREADS
#define CUIMAGE_REDUCE_THREADS 512
#endif

#define NBLOCKS(N,blockSize) (N/blockSize+(N%blockSize==0?0:1))
#define NBLOCKSL(N,blockSize) 1

__global__ void array_copy(float* __restrict__ a, float* __restrict__ c, size_t mx, size_t my, size_t mz, size_t sx, size_t sy, size_t sz, size_t ox, size_t oy, size_t oz)
{
    size_t pnum = blockIdx.x * blockDim.x + threadIdx.x; // Optimized linear index computation
    size_t num_elements = (sx/2) * (sy/2); // Calculate total number of elements

    if (pnum >= num_elements) return; // Early exit for threads without work

    size_t px = pnum % (sx/2); // Compute x position
    size_t py = pnum / (sx/2); // Compute y position

    size_t ids = 2 * (px + py * sx); // Source index
    size_t idd = 2 * ((ox + px) + (oy + py) * sx); // Destination index

    // Efficient swap of elements using local variables
    float tmpR = c[idd];
    float tmpI = c[idd + 1];
    c[idd] = a[ids];
    c[idd + 1] = a[ids + 1];
    a[ids] = tmpR;
    a[ids + 1] = tmpI;
}