#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_arr(float b, float * c, int N)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (idx < N) {
        c[idx] = b;  // Assign value b to the element
    }
}