#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _init_mat(float *m, float val, int len) {
    // Use shared memory to reduce global memory transaction
    __shared__ float shared_val;
    if (threadIdx.x == 0) {
        shared_val = val;  // Initialize shared memory
    }
    __syncthreads();  // Ensure all threads have the updated shared memory

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < len) {
        m[tid] = shared_val;  // Use shared memory instead of global variable
    }
}