#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _copy_mat(float *m, float* target, int len) {
    // Use shared memory for faster access
    extern __shared__ float s_data[];
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int lane_id = threadIdx.x;

    // Load data into shared memory
    if (tid < len) {
        s_data[lane_id] = m[tid];
    }
    __syncthreads();

    // Write data from shared memory to global memory
    if (tid < len) {
        target[tid] = s_data[lane_id];
    }
}