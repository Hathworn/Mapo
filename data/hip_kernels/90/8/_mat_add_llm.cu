#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _mat_add(float *ma, float *mb, float *target, float sa, float sb, int len) {
    // Calculate the global thread ID for indexing
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Process multiple elements per thread for coalesced access and less divergence
    int stride = blockDim.x * gridDim.x; 
    for (int i = tid; i < len; i += stride) {
        target[i] = sa * ma[i] + sb * mb[i];
    }
}