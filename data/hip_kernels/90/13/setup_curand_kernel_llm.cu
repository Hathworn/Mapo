#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_curand_kernel(hiprandState *state, int count) {
    int id = threadIdx.x + blockIdx.x * blockDim.x; // Adjusted to use blockDim.x for flexibility
    if (id < count) {
        hiprand_init(1234, id, 0, &state[id]);
    }
}