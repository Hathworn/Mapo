#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_curand_kernel(hiprandState *state, int count){
int id = threadIdx.x + blockIdx.x * 64;
if(id < count){
hiprand_init(1234, id, 0, &state[id]);
}
}