#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _mat_sum_col(float *m, float *target, int nrow, int ncol) {
    // Use shared memory to improve memory access efficiency
    extern __shared__ float shared_sum[];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < ncol) {
        float sum = 0.0f;

        for (int i = threadIdx.x; i < nrow; i += blockDim.x) {
            sum += m[i * ncol + tid];
        }

        shared_sum[threadIdx.x] = sum;
        __syncthreads(); // Synchronize threads within block

        // Reduce using shared memory
        if (threadIdx.x == 0) {
            float total_sum = 0.0f;
            for (int i = 0; i < blockDim.x; ++i) {
                total_sum += shared_sum[i];
            }
            target[tid] = total_sum;
        }
    }
}