#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _ele_scale(float *m, float *target, float scaler, int len) {
    // Calculate thread's global index.
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Process multiple elements per thread to utilize all available threads effectively.
    for (int i = tid; i < len; i += blockDim.x * gridDim.x) {
        target[i] = scaler * m[i];
    }
}