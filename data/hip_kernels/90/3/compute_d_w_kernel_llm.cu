#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_d_w_kernel(float *v, float *h, float *dw, bool is_init, int input_size, int lu_padding, int channel_num, int filter_num, int filter_size, int feature_map_size){

    int imgIdx = blockIdx.y / (feature_map_size / 32);
    int filterIdx = blockIdx.x / (channel_num * feature_map_size / 32);
    int channelIdx = (blockIdx.x % (channel_num * feature_map_size / 32)) / (feature_map_size / 32);
    int tx = (blockIdx.x % (channel_num * feature_map_size / 32)) % (feature_map_size / 32) * 32 + threadIdx.x;
    int ty = (blockIdx.y % (feature_map_size / 32)) * 32 + threadIdx.y;

    __shared__ float shV[32 + MAX_FILETER_SIZE][32 + MAX_FILETER_SIZE];
    __shared__ float shH[32][32];

    float sign = is_init ? 1.0f : -1.0f;
    
    v = v + imgIdx * channel_num * input_size * input_size + channelIdx * input_size * input_size;
    h = h + imgIdx * filter_num * feature_map_size * feature_map_size + filterIdx * feature_map_size * feature_map_size;
    dw = dw + filterIdx * channel_num * filter_size * filter_size + channelIdx * filter_size * filter_size;
    
    float local_dw = 0.0f;

    // Unroll loops to reduce loop overhead and improve memory access patterns
    #pragma unroll
    for(int loadY = 0; loadY < 32; loadY += filter_size) {
        #pragma unroll
        for(int loadX = 0; loadX < 32; loadX += filter_size) {
            
            if(loadY + threadIdx.y < 32 && loadX + threadIdx.x < 32) {
                shH[threadIdx.y + loadY][threadIdx.x + loadX] = h[(ty + loadY) * feature_map_size + (tx + loadX)];
            }

            bool inPadding = (tx + loadX < lu_padding || ty + loadY < lu_padding || tx + loadX >= (input_size + lu_padding) || ty + loadY >= (input_size + lu_padding));
            shV[threadIdx.y + loadY][threadIdx.x + loadX] = inPadding ? 0 : v[(ty + loadY - lu_padding) * input_size + (tx + loadX - lu_padding)];
        }
    }
    
    __syncthreads();
    
    // Reduce computation inside shared memory for better performance
    #pragma unroll
    for(int i = 0; i < 32; i++) {
        #pragma unroll
        for(int j = 0; j < 32; j++) {
            local_dw += shV[threadIdx.y + i][threadIdx.x + j] * shH[i][j];
        }
    }

    // Use atomic operations to avoid race conditions
    atomicAdd(dw + threadIdx.y * filter_size + threadIdx.x, sign * local_dw);
}