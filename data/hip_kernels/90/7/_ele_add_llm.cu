#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _ele_add(float *m, float *target, float val, int len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a stride loop for better global memory access pattern
    for (int i = tid; i < len; i += gridDim.x * blockDim.x) {
        target[i] = val + m[i];
    }
}