#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _mat_mul(float *ma, float *mb, float *target, int len){
    // Using shared memory for better performance
    extern __shared__ float shared_ma[];
    extern __shared__ float shared_mb[];
  
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    if(tid < len){
        // Load data into shared memory
        shared_ma[local_tid] = ma[tid];
        shared_mb[local_tid] = mb[tid];

        __syncthreads();

        // Perform multiplication using shared memory
        target[tid] = shared_ma[local_tid] * shared_mb[local_tid];
    }
}