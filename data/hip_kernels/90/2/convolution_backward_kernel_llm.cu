#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolution_backward_kernel(float *y_h, float *filters, float *vbias, float *target, float *y_v, int input_size, int lu_padding, int channel_num, int feature_map_size, int filter_num, int filter_size, float *rnd_array, int rnd_num) {
    // Calculate global index
    int imgIdx = blockIdx.y / (input_size / 16);
    int channelIdx = blockIdx.x / (input_size / 16);
    int tx = (blockIdx.x % (input_size / 16)) * 16 + threadIdx.x;
    int ty = (blockIdx.y % (input_size / 16)) * 16 + threadIdx.y;
    int padding = (filter_size - 1);

    // Compute random index and value
    int rnd_index = ((blockIdx.y * blockDim.y + threadIdx.y) * (blockIdx.x * blockDim.x)  + threadIdx.x) % rnd_num;
    float rnd = rnd_array[rnd_index];

    // Shared memory for part of input and filter
    __shared__ float shHidden[16+2*(MAX_FILETER_SIZE-1)][16+2*(MAX_FILETER_SIZE-1)];
    __shared__ float shFlipFilter[MAX_FILETER_SIZE][MAX_FILETER_SIZE];
    float local_target = 0.0f;

    // Offset pointers
    target = target + imgIdx * channel_num * input_size * input_size + channelIdx * input_size * input_size;
    float *target_y_v = y_v + imgIdx * channel_num * input_size * input_size + channelIdx * input_size * input_size;

    __syncthreads();

    // Loop over filters
    for(int f = 0; f < filter_num; f++) {
        float *cur_y_h = y_h + imgIdx * filter_num * feature_map_size * feature_map_size + f * feature_map_size * feature_map_size;
        float *cur_filter = filters + f * channel_num * filter_size * filter_size + channelIdx * filter_size * filter_size;

        // Load current filter into shared memory (flipped)
        if(threadIdx.x < filter_size && threadIdx.y < filter_size) {
            shFlipFilter[threadIdx.y][threadIdx.x] = cur_filter[(filter_size - 1 - threadIdx.y) * filter_size + filter_size - 1 - threadIdx.x];
        }

        // Load visible layer into shared memory
        float *shHiddenLoad = &shHidden[threadIdx.y][threadIdx.x];
        if(tx < padding || ty < padding) {
            *shHiddenLoad = 0;
        } else {
            *shHiddenLoad = cur_y_h[(ty-padding) * input_size + (tx-padding)];
        }

        // Load right part into shared memory if needed
        if(threadIdx.x < 2 * padding) {
            shHiddenLoad = &shHidden[threadIdx.y][threadIdx.x+16];
            if(ty < padding || (tx+16) >= (feature_map_size+padding)) {
                *shHiddenLoad = 0;
            } else {
                *shHiddenLoad = cur_y_h[(ty-padding) * feature_map_size + (tx+16-padding)];
            }
        }

        // Load bottom part into shared memory if needed
        if(threadIdx.y < 2 * padding) {
            shHiddenLoad = &shHidden[threadIdx.y+16][threadIdx.x];
            if(tx < padding || (ty+16) >= (feature_map_size+padding)) {
                *shHiddenLoad = 0;
            } else {
                *shHiddenLoad = cur_y_h[(ty+16-padding) * feature_map_size + (tx-padding)];
            }

            // Load bottom-right corner into shared memory if needed
            if(threadIdx.x < 2 * padding) {
                shHiddenLoad = &shHidden[threadIdx.y+16][threadIdx.x+16];
                if((ty+16) >= (feature_map_size+padding) || (tx+16) >= (feature_map_size+padding)) {
                    *shHiddenLoad = 0;
                } else {
                    *shHiddenLoad = cur_y_h[(ty+16-padding) * feature_map_size + (tx+16-padding)];
                }
            }
        }

        __syncthreads();

        // Convolution operation
        for(int i = 0; i < filter_size; i++) {
            for(int j = 0; j < filter_size; j++) {
                local_target += shHidden[threadIdx.y + i + lu_padding][threadIdx.x + j + lu_padding] * shFlipFilter[i][j];
            }
        }

        __syncthreads();
    }

    local_target += vbias[channelIdx];

    // Threshold operation
    if(rnd < local_target) {
        target_y_v[ty * input_size + tx] = 1;
    } else {
        target_y_v[ty * input_size + tx] = 0;
    }
    target[ty * input_size + tx] = local_target;
}