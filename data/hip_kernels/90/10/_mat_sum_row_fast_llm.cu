#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _mat_sum_row_fast(float *m, float *target, int nrow, int ncol, int agg_col) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use dynamic shared memory to size the array based on block size
    extern __shared__ float accum[];

    float sum = 0.0f;

    // Loop over columns to accumulate contributions
    for (int i = tx; i < ncol; i += blockDim.x * gridDim.x) {
        sum += m[blockIdx.y * ncol + i];
    }
    
    accum[threadIdx.x] = sum;
    __syncthreads();

    // Reduce within a block
    // Reduce shared memory in-place with a single warp
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            accum[threadIdx.x] += accum[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // The first thread writes the final result
    if (threadIdx.x == 0) {
        target[blockIdx.y * agg_col + blockIdx.x] = accum[0];
    }
}