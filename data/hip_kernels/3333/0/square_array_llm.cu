#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    // Calculate the thread index using block and grid size
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop through the array with stride to handle larger arrays
    for (int i = idx; i < N; i += stride)
    {
        a[i] = a[i] * a[i];
    }
}