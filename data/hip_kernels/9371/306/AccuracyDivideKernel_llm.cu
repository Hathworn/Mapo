#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Launch single thread to perform whole division operation
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}