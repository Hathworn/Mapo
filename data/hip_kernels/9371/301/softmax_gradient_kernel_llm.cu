#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    
    float sum = 0;
    // Unroll loop to improve performance.
    for (int i = idx; i < dim; i += blockDim.x) {
        sum += dY[i] * Y[i];
    }
    reduction_buffer[idx] = sum;
    __syncthreads();

    // Optimize reduction using binary tree.
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }

    sum = reduction_buffer[0];

    // Compute gradient, optimizing for memory coalescing.
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - sum);
    }
}