#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long idx = threadIdx.x + blockDim.x * blockIdx.x;
    idx += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
    if (idx >= no_elements) return;

    int base_ipidx = translate_idx_inv(idx, d1, d2, d3, scale_factor, 0, 0);
    float accumulated_grad = 0.0f; // Accumulator for gradient updates

    // Use loop unrolling to reduce loop overhead
    for (int i = 0; i < scale_factor; ++i) {
        for (int j = 0; j < scale_factor; ++j) {
            int off_ipidx = translate_idx_inv(0, d1, d2, d3, scale_factor, i, j) - base_ipidx; // Offset calculation
            accumulated_grad += gradOutput_data[base_ipidx + off_ipidx];
        }
    }
    gradInput_data[idx] += accumulated_grad;
}