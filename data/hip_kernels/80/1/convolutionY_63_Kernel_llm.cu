#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define KERNEL_RADIUS 31
#define KERNEL_LENGTH (2 * KERNEL_RADIUS + 1)

__constant__ float c_Kernel[KERNEL_LENGTH];

__global__ void convolutionY_63_Kernel(float *d_Dst, float *d_Src, int imageW, int imageH, int imageD, int outofbounds, float outofboundsvalue) {
    __shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS + 2 * COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + 1];

    const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
    const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
    const int baseZ = blockIdx.z;

    const int firstPixelInLine = (COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS - threadIdx.y) * imageW;
    const int lastPixelInLine = (imageH - baseY - 1) * imageW;

    d_Src += baseZ * imageH * imageW + baseY * imageW + baseX;
    d_Dst += baseZ * imageH * imageW + baseY * imageW + baseX;

    // Main data
    #pragma unroll
    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++) {
        int idx = threadIdx.y + i * COLUMNS_BLOCKDIM_Y;
        if (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) {
            s_Data[threadIdx.x][idx] = d_Src[i * COLUMNS_BLOCKDIM_Y * imageW];
        } else {
            if (outofbounds == 0) {
                s_Data[threadIdx.x][idx] = 0;
            } else if (outofbounds == 1) {
                s_Data[threadIdx.x][idx] = outofboundsvalue;
            } else {
                s_Data[threadIdx.x][idx] = d_Src[lastPixelInLine];
            }
        }
    }

    // Upper halo
    #pragma unroll
    for (int i = 0; i < COLUMNS_HALO_STEPS; i++) {
        int idx = threadIdx.y + i * COLUMNS_BLOCKDIM_Y;
        if (baseY >= -i * COLUMNS_BLOCKDIM_Y) {
            s_Data[threadIdx.x][idx] = d_Src[i * COLUMNS_BLOCKDIM_Y * imageW];
        } else {
            if (outofbounds == 0) {
                s_Data[threadIdx.x][idx] = 0;
            } else if (outofbounds == 1) {
                s_Data[threadIdx.x][idx] = outofboundsvalue;
            } else {
                s_Data[threadIdx.x][idx] = d_Src[firstPixelInLine];
            }
        }
    }

    // Lower halo
    #pragma unroll
    for (int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS; i++) {
        int idx = threadIdx.y + i * COLUMNS_BLOCKDIM_Y;
        if (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) {
            s_Data[threadIdx.x][idx] = d_Src[i * COLUMNS_BLOCKDIM_Y * imageW];
        } else {
            if (outofbounds == 0) {
                s_Data[threadIdx.x][idx] = 0;
            } else if (outofbounds == 1) {
                s_Data[threadIdx.x][idx] = outofboundsvalue;
            } else {
                s_Data[threadIdx.x][idx] = d_Src[lastPixelInLine];
            }
        }
    }

    __syncthreads();

    if (baseX >= imageW) return;

    #pragma unroll
    for (int i = COLUMNS_HALO_STEPS; i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++) {
        if (imageH - baseY > i * COLUMNS_BLOCKDIM_Y) {
            float sum = 0;
            #pragma unroll
            for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++) {
                sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];
            }
            d_Dst[i * COLUMNS_BLOCKDIM_Y * imageW] = sum;
        }
    }
}