#include "hip/hip_runtime.h"
```c++
#include "hip/hip_runtime.h"
#include "includes.h"

#define KERNEL_RADIUS 31
#define KERNEL_LENGTH (2 * KERNEL_RADIUS + 1)

__constant__ float c_Kernel[ KERNEL_LENGTH ];

__global__ void convolutionX_63_Kernel(float *d_Dst, float *d_Src, int imageW, int imageH, int imageD, int outofbounds, float outofboundsvalue)
{
    __shared__ float s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X];

    const int baseX = (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;
    const int baseZ = blockIdx.z;

    const int firstPixelInLine = ROWS_BLOCKDIM_X * ROWS_HALO_STEPS - threadIdx.x;
    const int lastPixelInLine = imageW - baseX - 1;

    // set the input and output arrays to the right offset
    d_Src += baseZ * imageH * imageW + baseY * imageW + baseX;
    d_Dst += baseZ * imageH * imageW + baseY * imageW + baseX;

    // Define out of bounds values
    const float outOfBoundsVal = (outofbounds == 0) ? 0.0f : ((outofbounds == 1) ? outofboundsvalue : d_Src[(imageW - 1 < baseX) ? lastPixelInLine : firstPixelInLine]);

    // Load main data and halos
    #pragma unroll
    for (int i = 0; i < ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS; i++)
    {
        int dataX = baseX + (i * ROWS_BLOCKDIM_X);
        if (dataX < 0 || dataX >= imageW)
        {
            s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = outOfBoundsVal;
        }
        else
        {
            s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] = d_Src[i * ROWS_BLOCKDIM_X];
        }
    }

    __syncthreads();

    // this pixel is not part of the image and does not need to be convolved
    if (baseY >= imageH)
        return;

    #pragma unroll
    for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++)
    {
        if (imageW - baseX > i * ROWS_BLOCKDIM_X)
        {
            float sum = 0;

            #pragma unroll
            for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++)
            {
                sum += c_Kernel[KERNEL_RADIUS - j] * s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j];
            }

            d_Dst[i * ROWS_BLOCKDIM_X] = sum;
        }
    }
}