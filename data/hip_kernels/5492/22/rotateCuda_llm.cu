#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rotateCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, float inXStart, float inYStart, uint32_t width, uint32_t height, float cosAngle, float sinAngle) {
    uint32_t outX = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t outY = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if the thread corresponds to a valid pixel
    if (outX < width && outY < height) {
        // Compute the input coordinates for the current output pixel
        const float exactInX = inXStart + cosAngle * outX + sinAngle * outY;
        const float exactInY = inYStart - sinAngle * outX + cosAngle * outY;

        const int32_t inX = static_cast<int32_t>(exactInX);
        const int32_t inY = static_cast<int32_t>(exactInY);

        // Calculate the location of the output pixel
        uint8_t *outPixel = out + outY * rowSizeOut + outX;

        // Validate input coordinates
        if (inX >= 0 && inX < static_cast<int32_t>(width) - 1 && inY >= 0 && inY < static_cast<int32_t>(height) - 1) {
            // Calculate the position of the input pixel
            const uint8_t *inPixel = in + inY * rowSizeIn + inX;

            // Perform bilinear interpolation
            const float probX = exactInX - inX;
            const float probY = exactInY - inY;
            const float mean = *inPixel * (1 - probX) * (1 - probY) +
                               *(inPixel + 1) * probX * (1 - probY) +
                               *(inPixel + rowSizeIn) * (1 - probX) * probY +
                               *(inPixel + rowSizeIn + 1) * probX * probY +
                               0.5f;

            *outPixel = static_cast<uint8_t>(mean);
        } else {
            *outPixel = 0; // Default to 0 for out of bounds
        }
    }
}