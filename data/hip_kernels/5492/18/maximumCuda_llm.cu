#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void maximumCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate global index once per thread
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure index doesn't exceed bounds
    if (x < width && y < height) {
        uint32_t indexIn1 = y * rowSizeIn1 + x;
        uint32_t indexIn2 = y * rowSizeIn2 + x;
        uint32_t indexOut = y * rowSizeOut + x;

        // Use max function for clarity
        out[indexOut] = max(in1[indexIn1], in2[indexIn2]);
    }
}