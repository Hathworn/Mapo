#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogramCuda(const uint8_t *data, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t *histogram)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Use shared memory for histogram to reduce global memory accesses
    __shared__ uint32_t localHistogram[256];
    
    // Initialize shared histogram
    if (threadIdx.x < 256)
        localHistogram[threadIdx.x] = 0;
    __syncthreads();

    if (x < width && y < height)
    {
        const uint32_t id = y * rowSize + x;
        atomicAdd(&localHistogram[data[id]], 1);
    }
    __syncthreads();

    // Reduce results to global histogram
    if (threadIdx.x < 256)
        atomicAdd(&histogram[threadIdx.x], localHistogram[threadIdx.x]);
}