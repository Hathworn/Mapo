#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flipCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height, bool horizontal, bool vertical) {
    // Cache thread indexes
    const uint32_t inX = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t inY = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Check boundaries
    if (inX < width && inY < height) {
        // Precompute output indexes to reduce computation
        const uint32_t outX = horizontal ? (width - 1 - inX) : inX;
        const uint32_t outY = vertical ? (height - 1 - inY) : inY;
        
        // Optimized memory access in a single step
        out[outY * rowSizeOut + outX] = in[inY * rowSizeIn + inX];
    }
}