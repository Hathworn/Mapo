#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void projectionProfileVerticalCuda(const uint8_t *image, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t *projection)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Use shared memory to reduce global memory access and atomic operations.
    __shared__ uint32_t sharedProjection[1024];

    if(threadIdx.x == 0 && y < height) {
        sharedProjection[y] = 0;
    }
    __syncthreads();

    if (x < width && y < height)
    {
        const uint8_t *imageY = image + y * rowSize + x;
        atomicAdd(&sharedProjection[y], (*imageY));
    }
    __syncthreads();

    if(threadIdx.x == 0 && y < height) {
        atomicAdd(&projection[y], sharedProjection[y]);
    }
}