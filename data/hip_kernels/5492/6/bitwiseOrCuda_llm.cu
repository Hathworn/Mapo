#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitwiseOrCuda(const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Pre-calculate block and thread positions
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Compute single index for rowSize calculations
    const uint32_t index = y * rowSizeIn1 + x; 

    // Use thread-safe boundary checks and avoid unnecessary calculations
    if (x < width && y < height)
    {
        out[y * rowSizeOut + x] = in1[index] | in2[y * rowSizeIn2 + x];
    }
}