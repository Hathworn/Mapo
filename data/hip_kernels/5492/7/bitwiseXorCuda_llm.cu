#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitwiseXorCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate global thread ID
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure thread is within image bounds
    if (x < width && y < height) {
        // Compute linear indices directly for thread
        out[y * rowSizeOut + x] = in1[y * rowSizeIn1 + x] ^ in2[y * rowSizeIn2 + x];
    }
}