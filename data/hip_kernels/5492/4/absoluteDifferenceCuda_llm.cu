#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void absoluteDifferenceCuda(const uint8_t * __restrict__ in1, uint32_t rowSizeIn1, const uint8_t * __restrict__ in2, uint32_t rowSizeIn2, uint8_t * __restrict__ out, uint32_t rowSizeOut, uint32_t width, uint32_t height) 
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure thread is within bounds
    if (x < width && y < height) {
        // Calculate linear index
        const uint32_t idx1 = y * rowSizeIn1 + x;
        const uint32_t idx2 = y * rowSizeIn2 + x;
        const uint32_t idxOut = y * rowSizeOut + x;

        // Load input values and calculate the absolute difference
        const uint8_t val1 = in1[idx1];
        const uint8_t val2 = in2[idx2];
        out[idxOut] = (val1 > val2) ? (val1 - val2) : (val2 - val1);
    }
}