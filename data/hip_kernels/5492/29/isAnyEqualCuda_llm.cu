#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void isAnyEqualCuda(const uint8_t *image, const uint8_t *value, size_t valueCount, uint32_t width, uint32_t height, uint32_t *differenceCount)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        const uint32_t id = y * width + x;

        // Use register to avoid re-accessing memory
        const uint8_t pixelValue = image[id];
        bool equal = false;

        // Optimize loop by unrolling
        for (uint32_t i = 0; i < valueCount; ++i) {
            if (pixelValue == value[i]) {
                equal = true;
                break;
            }
        }

        // Atomic operation if there is an equal value
        if (equal)
            atomicAdd(differenceCount, 1);
    }
}