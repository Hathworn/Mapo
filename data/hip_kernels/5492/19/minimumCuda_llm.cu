#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void minimumCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate global index once
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        // Combine operations to avoid multiple memory accesses
        const uint32_t indexIn1 = y * rowSizeIn1 + x;
        const uint32_t indexIn2 = y * rowSizeIn2 + x;
        const uint32_t indexOut = y * rowSizeOut + x;

        // Directly perform minimum computation
        out[indexOut] = min(in1[indexIn1], in2[indexIn2]);
    }
}