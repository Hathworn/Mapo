#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void projectionProfileHorizontalCuda(const uint8_t * image, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * projection)
{
    // Calculate global thread index
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; 

    if (y < height) {
        // Each thread in this block processes multiple elements in a loop
        for (uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; x < width; x += blockDim.x * gridDim.x) {
            const uint8_t * imageX = image + y * rowSize + x;
            atomicAdd(&projection[x], (*imageX));
        }
    }
}