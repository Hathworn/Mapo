#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height) {
    // Calculate the global thread index for this dimension.
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread is within bounds before copying.
    if (x < width && y < height) {
        // Use efficient memory copy for each pixel.
        out[y * rowSizeOut + x] = in[y * rowSizeIn + x];
    }
}