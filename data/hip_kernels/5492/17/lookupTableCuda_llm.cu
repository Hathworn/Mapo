#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lookupTableCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height, uint8_t *table)
{
    // Calculate global thread index for 2D grid
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Use shared memory to reduce global memory access
    __shared__ uint8_t localTable[256];  // Assuming table size is known as 256
    
    // Load table into shared memory only by the first thread once
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        for (int i = 0; i < 256; i++) {
            localTable[i] = table[i];
        }
    }
    __syncthreads();  // Ensure all threads have the table data

    // Perform the lookup operation
    if (x < width && y < height) {
        out[y * rowSizeOut + x] = localTable[in[y * rowSizeIn + x]];
    }
}