#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subtractCuda(const uint8_t* __restrict__ in1, uint32_t rowSizeIn1, const uint8_t* __restrict__ in2, uint32_t rowSizeIn2, uint8_t* __restrict__ out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate global thread indexes
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if within image bounds
    if (x < width && y < height) {
        // Use pointer arithmetic for indices
        const uint8_t in1Val = in1[y * rowSizeIn1 + x];
        const uint8_t in2Val = in2[y * rowSizeIn2 + x];

        // Compute subtraction and store in output
        uint8_t* outX = out + y * rowSizeOut + x;
        *outX = (in1Val > in2Val) ? (in1Val - in2Val) : 0;
    }
}