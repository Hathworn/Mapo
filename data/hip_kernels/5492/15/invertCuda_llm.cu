#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void invertCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height) {
    // Use shared memory to optimize memory access
    extern __shared__ uint8_t sharedMem[];

    uint32_t localX = threadIdx.x;
    uint32_t localY = threadIdx.y;
    const uint32_t x = blockDim.x * blockIdx.x + localX;
    const uint32_t y = blockDim.y * blockIdx.y + localY;

    if (x < width && y < height) {
        // Load data to shared memory
        sharedMem[localY * blockDim.x + localX] = in[y * rowSizeIn + x];
        __syncthreads();

        // Perform inversion using shared memory
        out[y * rowSizeOut + x] = ~sharedMem[localY * blockDim.x + localX];
    }
}