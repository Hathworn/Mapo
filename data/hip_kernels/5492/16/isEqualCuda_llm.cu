#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void isEqualCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint32_t width, uint32_t height, uint32_t *isEqual)
{
    // Calculate global thread index
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure index is within bounds
    if (x < width && y < height) {
        // Perform comparison and update result atomically
        const uint32_t partsEqual = (in1[y * rowSizeIn1 + x] == in2[y * rowSizeIn2 + x]) ? 1 : 0;
        atomicAnd(isEqual, partsEqual);
    }
}