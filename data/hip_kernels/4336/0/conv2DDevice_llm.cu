#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TRUE (1)
#define FALSE (0)
#define NUM_OF_GLOBAL_ROWS (16000)
#define NUM_OF_GLOBAL_COLS (16000)
#define BLOCK_SIZE (32)
#define KERNEL_SIZE (2)
#define NUM_OF_LOCAL_ROWS (BLOCK_SIZE + (2 * KERNEL_SIZE))
#define NUM_OF_LOCAL_COLS (BLOCK_SIZE + (2 * KERNEL_SIZE))

typedef enum Status_Tag
{
PASSED,
FAILED
}Status_T;

typedef enum Cuda_Event_Tag
{
ALLOCATE_DEVICE_MATRIX_A,
ALLOCATE_DEVICE_MATRIX_B,
COPY_MATRIX_A_FROM_HOST_TO_DEVICE,
LAUNCH_KERNEL_CONV2DDEVICE,
DEVICE_SYNCHRONIZATION,
COPY_MATRIX_B_FROM_DEVICE_TO_HOST,
FREE_DEVICE_MATRIX_A,
FREE_DEVICE_MATRIX_B
}Cuda_Event_T;

typedef enum Corner_Cell_Name_Tag
{
LEFT_TOP_PADDING_CORNER,
RIGHT_TOP_PADDING_CORNER,
LEFT_BOTTOM_PADDING_CORNER,
RIGHT_BOTTOM_PADDING_CORNER,
NUM_OF_CORNERS
}Corner_Cell_Name_T;

typedef enum Ver_Side_Cell_Name_Tag
{
LEFT_PADDING_CELL,
RIGHT_PADDING_CELL,
NUM_OF_VER_SIDES
}Ver_Side_Cell_Name_T;

typedef enum Hor_Side_Cell_Name_Tag
{
TOP_PADDING_CELL,
BOTTOM_PADDING_CELL,
NUM_OF_HOR_SIDES
}Hor_Side_Cell_Name_T;

typedef struct Result_Tag
{
Status_T status;
int index;
}Result_T;

typedef struct Cell_Tag
{
int r_idx;
int c_idx;
}Cell_T;

#define MATRIX_TO_ARRAY_INDEX(r_idx, c_idx, num_cols) ((r_idx*num_cols) + c_idx)

__global__ void conv2DDevice(const int *in, int *out)
{
    int g_col_idx = blockDim.x * blockIdx.x + threadIdx.x;
    int g_row_idx = blockDim.y * blockIdx.y + threadIdx.y;
    int l_col_idx = threadIdx.x + KERNEL_SIZE;
    int l_row_idx = threadIdx.y + KERNEL_SIZE;

    __shared__ int local[NUM_OF_LOCAL_ROWS * NUM_OF_LOCAL_COLS];

    int g_idx = MATRIX_TO_ARRAY_INDEX(g_row_idx, g_col_idx, NUM_OF_GLOBAL_COLS);
    int l_idx = MATRIX_TO_ARRAY_INDEX(l_row_idx, l_col_idx, NUM_OF_LOCAL_COLS);

    if ((g_row_idx < NUM_OF_GLOBAL_ROWS) && (g_col_idx < NUM_OF_GLOBAL_COLS))
    {
        // Load central region into shared memory
        local[l_idx] = in[g_idx];

        // Load left and right padding
        if (threadIdx.x < KERNEL_SIZE)
        {
            if (g_col_idx >= KERNEL_SIZE) {
                local[MATRIX_TO_ARRAY_INDEX(l_row_idx, l_col_idx - KERNEL_SIZE, NUM_OF_LOCAL_COLS)] = in[g_idx - KERNEL_SIZE];
            } else {
                local[MATRIX_TO_ARRAY_INDEX(l_row_idx, l_col_idx - KERNEL_SIZE, NUM_OF_LOCAL_COLS)] = 0;
            }

            if (g_col_idx + BLOCK_SIZE < NUM_OF_GLOBAL_COLS) {
                local[MATRIX_TO_ARRAY_INDEX(l_row_idx, l_col_idx + BLOCK_SIZE, NUM_OF_LOCAL_COLS)] = in[g_idx + BLOCK_SIZE];
            } else {
                local[MATRIX_TO_ARRAY_INDEX(l_row_idx, l_col_idx + BLOCK_SIZE, NUM_OF_LOCAL_COLS)] = 0;
            }
        }

        // Load top and bottom padding
        if (threadIdx.y < KERNEL_SIZE)
        {
            if (g_row_idx >= KERNEL_SIZE) {
                local[MATRIX_TO_ARRAY_INDEX(l_row_idx - KERNEL_SIZE, l_col_idx, NUM_OF_LOCAL_COLS)] = in[g_idx - (KERNEL_SIZE * NUM_OF_GLOBAL_COLS)];
            } else {
                local[MATRIX_TO_ARRAY_INDEX(l_row_idx - KERNEL_SIZE, l_col_idx, NUM_OF_LOCAL_COLS)] = 0;
            }

            if (g_row_idx + BLOCK_SIZE < NUM_OF_GLOBAL_ROWS) {
                local[MATRIX_TO_ARRAY_INDEX(l_row_idx + BLOCK_SIZE, l_col_idx, NUM_OF_LOCAL_COLS)] = in[g_idx + (BLOCK_SIZE * NUM_OF_GLOBAL_COLS)];
            } else {
                local[MATRIX_TO_ARRAY_INDEX(l_row_idx + BLOCK_SIZE, l_col_idx, NUM_OF_LOCAL_COLS)] = 0;
            }
        }

        // Load corners
        if ((threadIdx.x < KERNEL_SIZE) && (threadIdx.y < KERNEL_SIZE))
        {
            // preload and pre-zero all corners
            int corners[4] = {0};
            
            int x_offsets[4] = {-KERNEL_SIZE, BLOCK_SIZE, -KERNEL_SIZE, BLOCK_SIZE};
            int y_offsets[4] = {-KERNEL_SIZE, -KERNEL_SIZE, BLOCK_SIZE, BLOCK_SIZE};

            for (int i = 0; i < 4; ++i) {
                int c_row_idx = g_row_idx + y_offsets[i];
                int c_col_idx = g_col_idx + x_offsets[i];
                if (c_row_idx >= 0 && c_col_idx >= 0 && c_row_idx < NUM_OF_GLOBAL_ROWS && c_col_idx < NUM_OF_GLOBAL_COLS) {
                    corners[i] = in[g_idx + (y_offsets[i] * NUM_OF_GLOBAL_COLS) + x_offsets[i]];
                }
                int c_local_idx = MATRIX_TO_ARRAY_INDEX(l_row_idx + y_offsets[i], l_col_idx + x_offsets[i], NUM_OF_LOCAL_COLS);
                local[c_local_idx] = corners[i];
            }
        }
    }

    __syncthreads();

    // Convolution result accumulation
    int result = 0;
    for (int row_offset = -KERNEL_SIZE; row_offset <= KERNEL_SIZE; ++row_offset)
    {
        for (int col_offset = -KERNEL_SIZE; col_offset <= KERNEL_SIZE; ++col_offset)
        {
            int l_ele_idx = MATRIX_TO_ARRAY_INDEX((l_row_idx + row_offset), (l_col_idx + col_offset), NUM_OF_LOCAL_COLS);
            result += local[l_ele_idx];
        }
    }

    out[g_idx] = result;
}