#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDoEigen(double* m, int rows, int columns)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global index
    if (idx < 4) // Ensure within bounds
    {
        printf("CUDA ptr: %p\n", m + idx);
        printf("CUDA value: %lf\n", m[idx]);
    }
}