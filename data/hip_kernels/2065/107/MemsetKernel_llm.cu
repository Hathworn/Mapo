#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global thread index for 1D flattened 2D grid
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single index to reduce computation
    int pos = i * w + j;

    // Check bounds in a single condition
    if (pos < w * h) {
        image[pos] = value;
    }
}