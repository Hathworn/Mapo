#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use if statement with common repeat code to reduce branching 
    if (ptidx >= npoints) return;

    // Fetch loc once to reduce memory access time
    short2 loc = loc_[ptidx];

    // Use variables to avoid repeated index calculations
    float scaledX = loc.x * scale;
    float scaledY = loc.y * scale;
    
    x[ptidx] = scaledX;
    y[ptidx] = scaledY;
}