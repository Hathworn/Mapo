#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst) {
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Precompute common expressions
    float value = src[image_row_offset + j];
    float cx = u[flow_row_offset + j] * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + static_cast<float>(i) + 1.0f;

    float px, py, dx = modff(cx, &px), dy = modff(cy, &py);

    int base_tx = static_cast<int>(px), base_ty = static_cast<int>(py);

    // Helper lambda for updating pixel
    auto update_pixel = [&](int tx, int ty, float w) {
        if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
            int offset = ty * image_stride + tx;
            atomicAdd(dst + offset, value * w); // Use HIP atomic functions
            atomicAdd(normalization_factor + offset, w);
        }
    };

    // Fill four corners
    update_pixel(base_tx, base_ty, dx * dy);
    update_pixel(base_tx - 1, base_ty, (1.0f - dx) * dy);
    update_pixel(base_tx - 1, base_ty - 1, (1.0f - dx) * (1.0f - dy));
    update_pixel(base_tx, base_ty - 1, dx * (1.0f - dy));
}