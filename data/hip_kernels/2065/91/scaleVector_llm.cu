#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Calculate global thread index
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Using grid-stride loop to handle large vectors
    for (int idx = pos; idx < len; idx += blockDim.x * gridDim.x)
    {
        d_res[idx] = d_src[idx] * scale;
    }
}