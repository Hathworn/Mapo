#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU(void)
{
    // Use a smaller warp size for the condition to reduce divergence
    int idx = threadIdx.x;
    
    // Limit thread execution to only relevant threads
    if (idx < 32 && idx == 5)
    {
        printf("Hello World from GPU thread %d!\n", idx);
    }
}