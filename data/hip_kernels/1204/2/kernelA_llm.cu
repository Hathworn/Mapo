#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelA(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    if (index < n) { // Optimize by checking bounds before loop
        for (int i = index; i < n; i += stride) {
            float xi = x[i]; // Cache x[i] into register to optimize memory access
            float yi = y[i]; // Cache y[i] into register to optimize memory access
            if (xi > yi) {
                float sum = 0.0f; // Accumulate result locally
                for (int j = 0; j < n / CONST; j++) {
                    sum += x[j] + y[j];
                }
                y[i] = sum; // Write result once
            } else {
                float divSum = 0.0f; // Accumulate result locally
                for (int j = 0; j < n / CONST; j++) {
                    divSum += x[j] / y[j];
                }
                y[i] = divSum; // Write result once
            }
        }
    }
}