#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *array, int size) {
    // Using a single thread to print the array size
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("### Array size: %d\n", size);
    }

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process only valid threads
    if (index < size) {
        array[index] += 1.f;
    }
}