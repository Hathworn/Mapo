#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelC(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Loop until n with step size equal to stride
    for (int i = index; i < n; i += stride) {
        // Check the condition and process only when it's true to avoid unnecessary computations
        if (x[i] > y[i]) {
            float sum = 0.0f;
            // Pre-calculate sum to avoid repeated memory access
            for (int j = 0; j < n / CONST; j++) {
                sum += x[j] + y[j];
            }
            y[i] = sum;  // Store pre-calculated sum
        }
    }
}