#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelB(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        float xi = x[i]; // Load x[i] once to reduce global memory access
        float yi = y[i]; // Load y[i] once to reduce global memory access
        if (xi > yi) {
            float sum = 0.0f; // Use a local variable to accumulate sum
            for (int j = 0; j < n / CONST; j++) {
                sum += x[j] + y[j];
            }
            y[i] = sum; // Update y[i] once outside the loop
        } else {
            yi = atomicAdd(&y[i], xi); // Use local yi for atomic operation
        }
    }
}
```
