#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelD(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use the shared memory for atomicAdd optimization
    extern __shared__ float shared_y[];

    // Load shared memory
    for (int j = threadIdx.x; j < n / CONST; j += blockDim.x) {
        shared_y[j] = y[j];
    }
    __syncthreads();

    // Perform atomicAdd operation in shared memory
    for (int i = index; i < n; i += stride) {
        for (int j = 0; j < n / CONST; j++) {
            atomicAdd(&shared_y[j], x[j]);
        }
    }
    __syncthreads();

    // Write back to global memory
    for (int j = threadIdx.x; j < n / CONST; j += blockDim.x) {
        y[j] = shared_y[j];
    }
}