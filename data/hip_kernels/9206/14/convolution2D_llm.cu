#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolution2D(const float *d_arr, const float *d_mask, float *d_result, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Perform boundary check only once
    if (i < N && j < N) {
        float avgSum = 0.0f;
        int cnum = 0;

        // Use loop unrolling for a 3x3 kernel
        #pragma unroll
        for (int p = i - 1; p <= i + 1; p++) {
            #pragma unroll
            for (int q = j - 1; q <= j + 1; q++) {
                if (p >= 0 && p < N && q >= 0 && q < N) {
                    avgSum += d_arr[p * N + q] * d_mask[cnum];
                }
                cnum++;
            }
        }

        // Store result directly outside loops
        d_result[i * N + j] = avgSum;
    }
}