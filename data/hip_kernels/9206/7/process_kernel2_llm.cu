#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void process_kernel2(const float* input, float* output, int numElements) {
    // Calculate a unique thread index within the grid
    int globalThreadId = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
                       + blockIdx.y * blockDim.x * blockDim.z
                       + blockIdx.z * blockDim.x
                       + threadIdx.z * blockDim.x * blockDim.y
                       + threadIdx.y * blockDim.x
                       + threadIdx.x;

    // Perform operation only for valid global thread IDs
    if(globalThreadId < numElements) {
        output[globalThreadId] = logf(fabsf(input[globalThreadId])); // Utilize logf and fabsf for performance improvements
    }
}