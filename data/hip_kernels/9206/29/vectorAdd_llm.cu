#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure that each thread computes only if within bounds
    if (i < numElements)
    {
        // Element-wise addition
        C[i] = A[i] + B[i];
    }
}