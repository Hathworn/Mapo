#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolution2D(const float *d_arr, const float *d_mask, float *d_result, int N) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    // Use shared memory to optimize memory access
    __shared__ float shared_arr[blockDim.x][blockDim.y];

    int globalId = i*N + j;
    if(i < N && j < N) {
        float avgSum = 0;
        int id, cnum = 0;

        // Load data into shared memory
        if (i < N && j < N) {
            shared_arr[threadIdx.x][threadIdx.y] = d_arr[globalId];
        }
        __syncthreads();

        for(int p = -1; p <= 1; p++) {
            for(int q = -1; q <= 1; q++) {
                int new_i = i + p;
                int new_j = j + q;
                if(new_i >= 0 && new_i < N && new_j >= 0 && new_j < N) {
                    // Access shared memory instead of global memory
                    avgSum += shared_arr[threadIdx.x + p][threadIdx.y + q] * d_mask[cnum];
                }
                cnum++;
            }
        }
        
        // Write result to global memory
        d_result[globalId] = avgSum;
    }
}