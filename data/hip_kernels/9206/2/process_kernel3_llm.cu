#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Device Function for process_kernel1


// Device Function for process_kernel2


// Device Function for process_kernel3

__global__ void process_kernel3(const float* input, float* output, int numElements) {

    // Calculate the unique thread ID directly using grid and block calculations
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x + 
                         (blockIdx.y * blockDim.y + threadIdx.y) * blockDim.x * gridDim.x + 
                         (blockIdx.z * blockDim.z + threadIdx.z) * blockDim.x * blockDim.y * gridDim.x * gridDim.y;

    // Check if globalThreadId is within the range of numElements
    if (globalThreadId < numElements) {
        // Compute the square root and store it in the output array
        output[globalThreadId] = sqrtf(input[globalThreadId]);
    }
}