#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReduceRowMajor2(int *g_idata, int *g_odata, int size) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tid = threadIdx.x;
    extern __shared__ int sdata[];

    // Initialize shared memory
    sdata[tid] = (i < size) ? g_idata[i] : 0;
    __syncthreads();

    // Perform unrolled reduction
    for (unsigned int s = 4; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) {
            // Fully unroll loop for s = 4
            int idx1 = index + s;
            int idx2 = index + s + 1;
            int idx3 = index + s + 2;
            int idx4 = index + s + 3;
            
            sdata[index] += sdata[idx1];
            sdata[index + 1] += sdata[idx2];
            sdata[index + 2] += sdata[idx3];
            sdata[index + 3] += sdata[idx4];
        }
        __syncthreads();
    }

    // Write result to global memory
    if (tid == 0) {
        g_odata[blockIdx.x * 4] = sdata[0];
        g_odata[blockIdx.x * 4 + 1] = sdata[1];
        g_odata[blockIdx.x * 4 + 2] = sdata[2];
        g_odata[blockIdx.x * 4 + 3] = sdata[3];
    }
}