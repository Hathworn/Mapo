#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void process_kernel3(const float* input, float* output, int numElements) {
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;  // Optimize global thread ID calculation
    int stride = blockDim.x * gridDim.x;  // Calculate stride for grid-stride loop

    for (int i = globalThreadId; i < numElements; i += stride) { // Use grid-stride loop for better parallelism
        output[i] = sqrtf(input[i]);  // Use single precision sqrt function
    }
}