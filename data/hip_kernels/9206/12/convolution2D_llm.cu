#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolution2D(const float *d_arr, const float *d_mask, float *d_result, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Optimize: Calculate row index using block and thread.
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Optimize: Calculate column index using block and thread.
    int globalId = i * N + j;

    if (i < N && j < N) {
        float avgSum = 0;
        int cnum = 0;
        for (int p = i - 1; p <= i + 1; p++) {
            for (int q = j - 1; q <= j + 1; q++) {
                // Optimize: Combine boundary checks in a single condition.
                if (p >= 0 && p < N && q >= 0 && q < N) {
                    avgSum += d_arr[p * N + q] * d_mask[cnum];
                }
                cnum++;
            }
        }
        d_result[globalId] = avgSum;
    }
}