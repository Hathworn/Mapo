#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Device function for warp reduction
__device__ void warpReduce(volatile int* sdata, int tid, int n) {
    if (tid + 32 < n) sdata[tid] += sdata[tid + 32];
    if (tid + 16 < n) sdata[tid] += sdata[tid + 16];
    if (tid + 8 < n) sdata[tid] += sdata[tid + 8];
    if (tid + 4 < n) sdata[tid] += sdata[tid + 4];
}

// Global kernel function optimized
__global__ void ReduceRowMajor5(int *g_idata, int *g_odata, int size) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tid = threadIdx.x;
    extern __shared__ int sdata[];

    // Load data into shared memory
    sdata[tid] = (i < size) ? g_idata[i] : 0;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s >= 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Perform warp reduction
    if (tid < 32) {
        warpReduce(sdata, tid, blockDim.x);
    }

    // Write result for this block to global memory
    if (tid == 0) {
        for (int j = 0; j < 4 && j < size; j++) {
            g_odata[blockIdx.x * 4 + j] = sdata[j];
        }
    }
}