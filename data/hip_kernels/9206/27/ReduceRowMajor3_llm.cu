#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ReduceRowMajor3(int *g_idata, int *g_odata, int size) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tid = threadIdx.x;
    extern __shared__ int sdata[];
    sdata[tid] = (i < size) ? g_idata[i] : 0; // Use ternary operator for simplicity
    __syncthreads();
    
    for (unsigned int s = blockDim.x / 2; s > 3; s >>= 1) { // Use bitwise operator for division by 2
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid < 4) { // Assign first 4 elements only, avoids unnecessary checks
        g_odata[blockIdx.x * 4 + tid] = sdata[tid];
    }
}