#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Device Function for process_kernel1

// Device Function for process_kernel2

// Device Function for process_kernel3

__global__ void process_kernel2(const float* input, float* output, int numElements) {
    // Calculate global thread ID using 3D block and thread index.
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x +
                         (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x +
                         (blockIdx.z * blockDim.z + threadIdx.z) * (gridDim.x * gridDim.y * blockDim.x * blockDim.y);

    // Check within bounds and perform calculations
    if (globalThreadId < numElements) {
        output[globalThreadId] = (float)log(fabs(input[globalThreadId]));
    }
}