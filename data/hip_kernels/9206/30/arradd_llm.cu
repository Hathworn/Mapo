#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void arradd(const int *md, const int *nd, int *pd, int size) {
    int myid = blockDim.x * blockIdx.x + threadIdx.x;
    // Use shared memory for loading inputs
    __shared__ int mds[256];
    __shared__ int nds[256];
    
    if (myid < size) {
        // Load data into shared memory to reduce global memory accesses
        mds[threadIdx.x] = md[myid];
        nds[threadIdx.x] = nd[myid];
        __syncthreads(); // Ensure shared memory is loaded
        
        // Perform addition using shared memory
        pd[myid] = mds[threadIdx.x] + nds[threadIdx.x];
    }
}