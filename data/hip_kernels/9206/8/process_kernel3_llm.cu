#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void process_kernel3(const float* __restrict__ input, float* __restrict__ output, int numElements) {
    // Compute global thread ID using simpler calculation
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x + 
                         blockIdx.y * blockDim.y * gridDim.x + threadIdx.y * blockDim.x * gridDim.x + 
                         blockIdx.z * blockDim.z * gridDim.x * gridDim.y + threadIdx.z * blockDim.x * gridDim.x * gridDim.y;

    if (globalThreadId < numElements) {
        output[globalThreadId] = sqrtf(input[globalThreadId]); // Use sqrtf for single precision
    }
}