#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    // Efficiently compute global thread index
    int i = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;

    // Use loop unrolling if applicable
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
        
        if (i + hipBlockDim_x < numElements) // Double the work per thread
        {
            C[i + hipBlockDim_x] = A[i + hipBlockDim_x] + B[i + hipBlockDim_x];
        }
    }
}