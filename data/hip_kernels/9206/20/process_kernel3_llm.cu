#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void process_kernel3(const float* input, float* output, int numElements) {
    // Calculate global thread ID optimized for 3D grid and block
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x 
                       + (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x 
                       + (blockIdx.z * blockDim.z + threadIdx.z) * gridDim.x * gridDim.y * blockDim.x * blockDim.y;

    // Only process valid threads
    if (globalThreadId < numElements) {
        output[globalThreadId] = (float)sqrt(input[globalThreadId]);
    }
}