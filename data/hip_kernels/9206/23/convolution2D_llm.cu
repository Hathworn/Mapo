#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolution2D(const float *d_arr, const float *d_mask, float *d_result, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Use shared memory for d_mask and parts of d_arr for faster access
    __shared__ float s_mask[9];
    if (threadIdx.x < 9) {
        s_mask[threadIdx.x] = d_mask[threadIdx.x];
    }
    __syncthreads();

    if (i < N && j < N) {
        float avgSum = 0;
        int id, cnum = 0;
        for (int p = i - 1; p <= i + 1; p++) {
            for (int q = j - 1; q <= j + 1; q++) {
                if (p >= 0 && p < N && q >= 0 && q < N) {
                    id = p * N + q;
                    avgSum += d_arr[id] * s_mask[cnum]; // Use shared memory mask
                }
                cnum++;
            }
        }
        d_result[i * N + j] = avgSum;
    }
}