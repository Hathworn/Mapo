#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void process_kernel2(const float* input, float* output, int numElements) {
    // Calculate global thread index more efficiently
    int globalThreadId = blockIdx.x * blockDim.x * blockDim.y * blockDim.z 
                        + blockIdx.y * blockDim.x * blockDim.z
                        + blockIdx.z * blockDim.z
                        + threadIdx.z * blockDim.x * blockDim.y 
                        + threadIdx.y * blockDim.x 
                        + threadIdx.x;

    if (globalThreadId < numElements) {
        output[globalThreadId] = logf(fabsf(input[globalThreadId])); // Use faster single-precision math operations
    }
}