#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define N 50
#define NewN 100

#define LifeN 500
#define numofthreads 512
int numofeles=0, capacity;

struct chromosome {
    long long weight = 0, value = 0;
    bool chromo[100003];
};
chromosome chromoele[N], *cudaChromo, *cudaNewpopulation, newpopulation[NewN], res, x[2];
int weight[100001], value[100001], *devValue, *devWeight, *devnumeles;

__global__ void initOne(chromosome *cudaChromo, const int numele, int *devValue, int *devWeight) {
    // Calculate unique thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure the thread index is within bounds
    if (idx < numele) {
        // Initialize chromosome if thread belongs to this block
        bool isInBlock = (blockIdx.x == idx % N);
        cudaChromo[blockIdx.x].chromo[idx % N] = isInBlock;

        // Set value and weight if initial position
        if (isInBlock) {
            cudaChromo[blockIdx.x].value = devValue[idx];
            cudaChromo[blockIdx.x].weight = devWeight[idx];
        }
    }
}