#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define N 50
#define NewN 100

#define LifeN 500
#define numofthreads 512
int numofeles=0,capacity;

struct chromosome
{
    long long weight=0, value=0;
    bool chromo[100003];
};
chromosome chromoele[N],*cudaChromo,*cudaNewpopulation,newpopulation[NewN],res,x[2];
int weight[100001],value[100001],*devValue,*devWeight,*devnumeles;

__global__ void hybrid(chromosome *cudaChromo, chromosome *cudaNewpopulation, int seed1, const int numele, int *devValue, int *devWeight)
{
    // Calculate global index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Limit processing within NewN elements
    if (idx < NewN) {
        hiprandState state;
        // Initialize the HIP random number generator
        hiprand_init(seed1, idx, 0, &state);
        
        // Generate random seeds for selection
        int seed2 = hiprand(&state) % N;
        int seed3 = hiprand(&state) % numele;

        // Copy chromosome data
        cudaNewpopulation[idx] = cudaChromo[idx % N];

        if (idx < NewN - N) {
            // Apply genetic operations for the main segment
            if (cudaNewpopulation[idx].chromo[seed3]) {
                // Reduce value and weight if chromo is true
                cudaNewpopulation[idx].value -= devValue[seed3];
                cudaNewpopulation[idx].weight -= devWeight[seed3];
            }
            // Change chromo based on a randomly selected chromosome
            cudaNewpopulation[idx].chromo[seed3] = cudaChromo[seed2].chromo[seed3];

            if (cudaNewpopulation[idx].chromo[seed3]) {
                // Increase value and weight if chromo is true after change
                cudaNewpopulation[idx].value += devValue[seed3];
                cudaNewpopulation[idx].weight += devWeight[seed3];
            }
        } else {
            // Perform mutation at the end segment
            bool currentChromo = cudaNewpopulation[idx].chromo[seed3];
            cudaNewpopulation[idx].chromo[seed3] = !currentChromo;

            // Adjust value and weight based on mutation
            int factor = cudaNewpopulation[idx].chromo[seed3] ? 1 : -1;
            cudaNewpopulation[idx].value += devValue[seed3] * factor;
            cudaNewpopulation[idx].weight += devWeight[seed3] * factor;
        }
    }
}