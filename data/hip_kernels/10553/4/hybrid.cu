#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define N 50
#define NewN 100

#define LifeN 500
#define numofthreads 512
int numofeles=0,capacity;

struct chromosome
{
long long weight=0, value=0;
bool chromo[100003];
};
chromosome chromoele[N],*cudaChromo,*cudaNewpopulation,newpopulation[NewN],res,x[2];
int weight[100001],value[100001],*devValue,*devWeight,*devnumeles;
__global__ void hybrid(chromosome *cudaChromo, chromosome *cudaNewpopulation, int seed1, const int numele, int *devValue, int *devWeight)
{

int idx = threadIdx.x + blockIdx.x*blockDim.x;
if (idx < NewN){
hiprandState state;
hiprand_init(seed1, idx, seed1, &state);
int seed2 = hiprand(&state) % N;
hiprand_init(seed1, idx, seed1, &state);
int seed3 = hiprand(&state) % numele;
cudaNewpopulation[idx] = cudaChromo[idx%N];

if (idx <NewN-N){

cudaNewpopulation[idx].value -= devValue[seed3] * (cudaNewpopulation[idx].chromo[seed3] ? 1 : 0);
cudaNewpopulation[idx].weight -= devWeight[seed3] * (cudaNewpopulation[idx].chromo[seed3] ? 1 : 0);
cudaNewpopulation[idx].chromo[seed3] = cudaChromo[seed2].chromo[seed3];
cudaNewpopulation[idx].value += devValue[seed3] * (cudaNewpopulation[idx].chromo[seed3] ? 1 : 0);
cudaNewpopulation[idx].weight += devWeight[seed3] * (cudaNewpopulation[idx].chromo[seed3] ? 1 : 0);
}
else{

cudaNewpopulation[idx].chromo[seed3] = cudaNewpopulation[idx].chromo[seed3] ? false : true;
//printf("\n%d\n", idx);
cudaNewpopulation[idx].value += devValue[seed3] *(cudaNewpopulation[idx].chromo[seed3]? 1 : -1);
cudaNewpopulation[idx].weight += devWeight[seed3] * (cudaNewpopulation[idx].chromo[seed3] ? 1 : -1);
}

}
}