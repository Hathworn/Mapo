#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define N 50
#define NewN 100

#define LifeN 500
#define numofthreads 512
int numofeles=0, capacity;

struct chromosome
{
    long long weight=0, value=0;
    bool chromo[100003];
};
chromosome chromoele[N], *cudaChromo, *cudaNewpopulation, newpopulation[NewN], res, x[2];
int weight[100001], value[100001], *devValue, *devWeight, *devnumeles;

__global__ void initpopulation(chromosome *cudaChromo, int seed, const int numofeles, int *devValue, int *devWeight)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Ensure index is within the valid range
    if (blockIdx.x < N && idx < numofeles) {
        hiprandState state;
        // Use unique thread index for seed initialization
        hiprand_init(seed, idx, 0, &state);
        // Directly assign random boolean
        cudaChromo[blockIdx.x].chromo[idx] = hiprand(&state) & 1;
    }
}