#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define N 50
#define NewN 100

#define LifeN 500
#define numofthreads 512
int numofeles=0, capacity;

struct chromosome {
    long long weight = 0, value = 0;
    bool chromo[100003];
};
chromosome chromoele[N], *cudaChromo, *cudaNewpopulation, newpopulation[NewN], res, x[2];
int weight[100001], value[100001], *devValue, *devWeight, *devnumeles;

__global__ void bitonic_sort_step(chromosome *cudanewpopulation, int j, int k)
{
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ixj = i ^ j; // Calculate partner index

    // Only process valid elements
    if (ixj > i) {
        bool isDescending = (i & k) != 0; // Determine sorting order

        // Swap elements if in the wrong order
        if ((isDescending ? (cudanewpopulation[i].value < cudanewpopulation[ixj].value)
                          : (cudanewpopulation[i].value > cudanewpopulation[ixj].value))) {
            // Exchange elements
            chromosome temp = cudanewpopulation[i];
            cudanewpopulation[i] = cudanewpopulation[ixj];
            cudanewpopulation[ixj] = temp;
        }
    }
}