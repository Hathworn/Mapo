#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define N 50
#define NewN 100

#define LifeN 500
#define numofthreads 512
int numofeles = 0, capacity;

struct chromosome
{
    long long weight = 0, value = 0;
    bool chromo[100003];
};

chromosome chromoele[N], *cudaChromo, *cudaNewpopulation, newpopulation[NewN], res, x[2];
int weight[100001], value[100001], *devValue, *devWeight, *devnumeles;

__global__ void evaluate(chromosome *cudaChromo, int *devValue, int *devWeight, int numele)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    
    if (idx < numele) // Ensure we do not exceed bounds
    {
        long long tempValue = 0;
        long long tempWeight = 0;

        #pragma unroll // Unroll loop for performance
        for (int i = 0; i < numele; i++)
        {
            if (cudaChromo[idx].chromo[i])
            {
                tempValue += devValue[i];
                tempWeight += devWeight[i];
            }
        }
        
        cudaChromo[idx].value = tempValue;
        cudaChromo[idx].weight = tempWeight;
    }
}