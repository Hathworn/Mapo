#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define PI 3.141592653589793
#define BLOCKSIZE 1024

__global__ void cuAdd(float *dst, const float * __restrict__ src, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        dst[id] += src[id]; // Utilize memory coalescing for improved access efficiency
    }
}