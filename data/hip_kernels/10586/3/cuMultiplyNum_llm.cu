#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define PI 3.141592653589793
#define BLOCKSIZE 1024

__global__ void cuMultiplyNum(float *dst, float num, int size)
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure within bounds, using warp-efficient conditional
    if (id < size) 
    {
        dst[id] *= num;
    }
}