#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addVector(int *d1_in, int *d2_in, int *d_out, int n) {
    // Calculate flattened array index using block and thread indices
    int ind = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure index is within bounds using more efficient memory access pattern
    if (ind < n) {
        // Utilize registers to optimize memory access for addition
        int temp1 = d1_in[ind];
        int temp2 = d2_in[ind];
        d_out[ind] = temp1 + temp2;
    }
}