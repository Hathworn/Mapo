#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256
using namespace std;

__global__ void calculate(int *arr_in, int* arr_out, int sz, int option) {
    int ind = threadIdx.x;
    int actual_ind = blockIdx.x * blockDim.x + ind;
    extern __shared__ int shared_mem[];

    // Load data into shared memory if within bounds
    if (actual_ind < sz) {
        shared_mem[ind] = arr_in[actual_ind];
    } else {
        if (option == 0 || option == 3)
            shared_mem[ind] = 0;
        else if (option == 1) // maximum
            shared_mem[ind] = -INT_MAX;
        else // minimum
            shared_mem[ind] = INT_MAX;
    }
    __syncthreads();

    // Perform reduction using shared memory
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (ind < i) {
            if (option == 0 || option == 3)
                shared_mem[ind] += shared_mem[ind + i];
            else if (option == 1) // maximum
                shared_mem[ind] = max(shared_mem[ind], shared_mem[ind + i]);
            else // minimum
                shared_mem[ind] = min(shared_mem[ind], shared_mem[ind + i]);
        }
        __syncthreads();
    }

    // Store the result for this block in global memory
    if (ind == 0) {
        arr_out[blockIdx.x] = shared_mem[0];
    }
}