#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multVector(int *d1_in, int *d2_in, int *d_out, int n, int m) {
    int ind = blockDim.x * blockIdx.x + threadIdx.x;
    if (ind < m) {
        int temp = 0;  // Use a temporary variable for accumulation
        for (int i = 0; i < n; i++) {
            temp += d1_in[i] * d2_in[i * m + ind];
        }
        d_out[ind] = temp;  // Write the result once
    }
}