#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multMatrix(int *d1_in, int *d2_in, int *d_out, int n, int m, int k) {
    // Calculate global row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds
    if (row < n && col < k) {
        int sum = 0;
        for (int i = 0; i < m; i++) {
            sum += d1_in[row * m + i] * d2_in[i * k + col];
        }
        d_out[row * k + col] = sum;
    }
}