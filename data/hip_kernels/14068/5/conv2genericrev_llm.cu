#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv2genericrev(float *input, float *kernel, float *output, int input_n, int input_h, int input_w, int kernel_n, int kernel_h, int kernel_w, float alpha, int stride_h, int stride_w)
{
    // Output dimensions
    int output_h = input_h - (kernel_h - 1) * stride_h;
    int output_w = input_w - (kernel_w - 1) * stride_w;

    // Block indices define which output this thread processes
    int kk = blockIdx.x;
    int ii = blockIdx.y;

    // Batch and kernel IDs
    int batch = threadIdx.z;
    int kid = threadIdx.x;
    int nkids = blockDim.x;

    // Compute thread ID and number of total threads
    int tid = kid + batch * blockDim.x;
    int nthreads = blockDim.x * blockDim.z;

    // Point to correct output position
    output += (kk * input_n + ii) * output_h * output_w;

    // Allocate shared memory for outputs
    extern __shared__ float shared_output[];

    // Initialize thread-local pointers and output position
    float *output_s = shared_output + tid * output_w * output_h;
    int yy = threadIdx.y;

    // Convolution loop
    float *output_p = output_s + yy * output_w;
    for (int xx = 0; xx < output_w; xx++) {
        // Initialize input and kernel pointers
        float *input_p = input + (ii + batch * input_n) * input_h * input_w + yy * stride_h * input_w + xx * stride_w;
        float *kernel_p = kernel + (kk + batch * kernel_n) * kernel_w * kernel_h;
        
        // Compute dot product
        float sum = 0.0f;
        for (int ky = 0; ky < kernel_h; ky++) {
            for (int kx = kid; kx < kernel_w; kx += nkids) {
                sum += input_p[kx] * kernel_p[kx];
            }
            input_p += input_w;
            kernel_p += kernel_w;
        }
        *(output_p++) = sum;
    }
    __syncthreads();

    // Reduction and writing back to global memory
    if (yy == 0) {
        // Reduction of outputs
        for (int k = 1; k < nthreads; k++) {
            for (int i = tid; i < output_w * output_h; i += nthreads) {
                shared_output[i] += shared_output[k * output_h * output_w + i];
            }
        }
        __syncthreads();

        // Update global output applying alpha
        for (int i = tid; i < output_w * output_h; i += nthreads) {
            output[i] += alpha * shared_output[i];
        }
    }
}