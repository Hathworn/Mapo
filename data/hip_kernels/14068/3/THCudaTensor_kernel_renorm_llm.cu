#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_renorm(float *data, const float value, const long size, const float maxnorm)
{
    __shared__ float buffer[32];
    long tx = threadIdx.x;
    long bx = blockIdx.x;
    long step = blockDim.x;
    float *row = data + size * bx;

    buffer[tx] = 0;

    // Unroll loop for efficiency
    for (long i = tx; i < size; i += step * 4)
    {
        float sum = 0;
        if (i < size) sum += powf(fabsf(row[i]), value);
        if (i + step < size) sum += powf(fabsf(row[i + step]), value);
        if (i + 2 * step < size) sum += powf(fabsf(row[i + 2 * step]), value);
        if (i + 3 * step < size) sum += powf(fabsf(row[i + 3 * step]), value);
        buffer[tx] += sum;
    }

    // Parallel reduction
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
    {
        __syncthreads();
        if (tx < stride)
        {
            buffer[tx] += buffer[tx + stride];
        }
    }

    // Clip norms
    __syncthreads();
    float norm = pow(buffer[0], 1 / value);
    if (norm > maxnorm)
    {
        norm = maxnorm / (norm + 1e-7f);
        // Renormalize
        for (long i = tx; i < size; i += step)
        {
            row[i] *= norm;
        }
    }
}