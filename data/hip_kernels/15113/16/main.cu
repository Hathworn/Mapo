#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "sampleMultinomialWithoutReplacement.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    hiprandStateMtgp32 *state = NULL;
hipMalloc(&state, XSIZE*YSIZE);
int totalSamples = 1;
int sample = 1;
float *dest = NULL;
hipMalloc(&dest, XSIZE*YSIZE);
long distributions = 1;
int categories = 1;
float *origDist = NULL;
hipMalloc(&origDist, XSIZE*YSIZE);
float *normDistPrefixSum = NULL;
hipMalloc(&normDistPrefixSum, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
sampleMultinomialWithoutReplacement<<<gridBlock, threadBlock>>>(state,totalSamples,sample,dest,distributions,categories,origDist,normDistPrefixSum);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
sampleMultinomialWithoutReplacement<<<gridBlock, threadBlock>>>(state,totalSamples,sample,dest,distributions,categories,origDist,normDistPrefixSum);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
sampleMultinomialWithoutReplacement<<<gridBlock, threadBlock>>>(state,totalSamples,sample,dest,distributions,categories,origDist,normDistPrefixSum);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}