#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;

        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithoutReplacement(hiprandStateMtgp32* state, int totalSamples, int sample, float* dest, long distributions, int categories, float* origDist, float* normDistPrefixSum) {
    for (long curDistBase = blockIdx.x * blockDim.y; curDistBase < distributions; curDistBase += gridDim.x * blockDim.y) {
        long curDist = curDistBase + threadIdx.y;

        // Only active threads update random numbers, reducing calls to hiprand_uniform
        float r = threadIdx.x == 0 && curDist < distributions ? hiprand_uniform(&state[blockIdx.x]) : 0.0f;

        // Synchronize within the warp to ensure r is broadcasted properly
        r = __shfl_sync(0xFFFFFFFF, r, 0);

        if (threadIdx.x == 0 && curDist < distributions) {
            int choice = binarySearchForMultinomial(
                normDistPrefixSum + curDist * categories,
                categories,
                r
            );
            dest[curDist * totalSamples + sample] = float(choice) + 1.0f;
            origDist[curDist * categories + choice] = 0.0f;
        }
    }
}