#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_copyUpperSymmetric(float *input, int n, int len)
{
    // Use shared memory to hold a block of input data
    extern __shared__ float sharedInput[];

    // Calculate the index and load the block into shared memory
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    const int totalThreads = gridDim.x * blockDim.x;
    
    for (; idx < len; idx += totalThreads) {
        const int r = idx % n;
        const int c = idx / n;

        // Load input data into shared memory to reduce global memory access
        sharedInput[threadIdx.x] = input[idx];
        __syncthreads();

        if (r > c) {
            input[idx] = sharedInput[r * n + c];
        }
        __syncthreads();
    }
}