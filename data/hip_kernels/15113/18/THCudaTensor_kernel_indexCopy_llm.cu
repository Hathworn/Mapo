#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexCopy(float *res, float *src, long *res_stride, float *index, long res_nDim, int dim, long idx_size, long src_size, long size_dim) 
{
    // Calculate the flattened thread index
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    long flat_size = src_size / idx_size;

    // Check if the thread index is within bounds
    if (thread_idx >= flat_size) return;

    // Use shared memory for index access
    __shared__ float shared_index[blockDim.x];
    if (threadIdx.x < idx_size) {
        shared_index[threadIdx.x] = index[threadIdx.x];
    }
    __syncthreads();

    // Iterate over index size and perform operations
    for (int i = 0; i < idx_size; i++) { 
        // Initialize residuals
        int leftover = thread_idx;
        long coeff = 0;
        int targetIdx = i * size_dim;
        int resIdx = 0;
        
        // Iterate over dimensions
        for (int d = 0; d < res_nDim; d++) {
            if (d == dim) {
                resIdx += ((long)(shared_index[i]) - 1) * res_stride[d]; // Use shared index
            } else {
                long stride_d = (d < dim) ? res_stride[d] / size_dim : res_stride[d];
                coeff = leftover / stride_d;
                leftover -= coeff * stride_d;
                targetIdx += coeff * res_stride[d];
                resIdx += coeff * res_stride[d];
            }
        }
        // Perform copy
        res[resIdx] = src[targetIdx];
    }
}