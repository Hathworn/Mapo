#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_indexFill(float *tensor, long* stride, float *index, long src_nDim, int dim, long idx_size, long tensor_size, long size_dim, float val)
{
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    long flat_size = tensor_size / idx_size;

    if (thread_idx < flat_size)
    {
        long coeff, leftover, srcIdx;

        // Loop unrolling to improve performance
        for (int i = 0; i < idx_size; i++)
        {
            leftover = thread_idx;
            srcIdx = 0;

            #pragma unroll
            for (int d = 0; d < src_nDim; d++)
            {
                if (d < dim) 
                {
                    coeff = leftover / (stride[d] / size_dim);  // Calculate coefficient
                    leftover -= coeff * (stride[d] / size_dim); // Update leftover
                    srcIdx += coeff * stride[d];                // Calculate srcIdx
                }
                else if (d > dim)
                {
                    coeff = leftover / stride[d];                 // Calculate coefficient
                    leftover -= coeff * stride[d];                // Update leftover
                    srcIdx += coeff * stride[d];                  // Calculate srcIdx
                }
            }
            tensor[srcIdx + (long)((index[i]) - 1) * stride[dim]] = val; // Set tensor value
        }
    }
}