#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_indexAdd(float *res, float *src, long* res_stride, float *index, long res_nDim, int dim, long idx_size, long src_size, long size_dim)
{
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    long flat_size = src_size / idx_size;

    // Ensure the thread_id is within bounds to prevent unnecessary execution.
    if (thread_idx < flat_size)
    {
        // Use shared memory to store res_stride which are used multiple times.
        extern __shared__ long shared_stride[];
        if (threadIdx.x < res_nDim)
        {
            shared_stride[threadIdx.x] = res_stride[threadIdx.x];
        }
        __syncthreads();

        long coeff = 0;
        for (int i = 0; i < idx_size; i++)
        {
            int leftover = thread_idx;
            int targetIdx = 0;
            int resIdx = 0;
            int stride_d_updated = shared_stride[dim] / size_dim;

            for (int d = 0; d < res_nDim; d++)
            {
                if (d < dim)
                {
                    long stride_d = stride_d_updated;
                    coeff = leftover / stride_d;
                    leftover -= coeff * stride_d;
                    targetIdx += coeff * stride_d * idx_size;
                    resIdx += coeff * shared_stride[d];
                }
                else if (d > dim)
                {
                    coeff = leftover / shared_stride[d];
                    leftover -= coeff * shared_stride[d];
                    targetIdx += coeff * shared_stride[d];
                    resIdx += coeff * shared_stride[d];
                }
            }
            // Use a local variable to hold the atomic addition result
            float src_value = src[targetIdx + i * shared_stride[dim]];
            atomicAdd(&res[resIdx + ((long)(index[i]) - 1) * shared_stride[dim]], src_value);
        }
    }
}