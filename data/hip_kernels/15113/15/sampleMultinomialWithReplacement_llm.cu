#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;

        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        // No probability mass or precision problems; just return the
        // first element
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithReplacement(hiprandStateMtgp32* state, int totalSamples, float* dest, long distributions, int categories, float* normDistPrefixSum) {
    for (long curDist = blockIdx.x; curDist < distributions; curDist += gridDim.x) {
        for (int sampleBase = 0; sampleBase < totalSamples; sampleBase += blockDim.y) {
            int sample = sampleBase + threadIdx.y;
            float r = hiprand_uniform(&state[blockIdx.x]);

            if (threadIdx.x == 0 && sample < totalSamples) {
                // Directly assign to destination with offset for indices
                dest[curDist * totalSamples + sample] = (float)(binarySearchForMultinomial(
                    normDistPrefixSum + curDist * categories, categories, r) + 1);
            }
        }
    }
}