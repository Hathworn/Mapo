#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexSelect_contiguous(float *tensor, float *src, long stride, float *index, long idxSize)
{
    // Use shared memory to optimize memory access patterns for better performance
    extern __shared__ float sharedIndex[];

    const int VT = 4;
    const int WARP_SIZE = 32;

    // Load index data into shared memory (only if idxSize is small enough to benefit)
    if (threadIdx.x < idxSize) {
        sharedIndex[threadIdx.x] = index[threadIdx.x];
    }
    __syncthreads(); // Ensure all threads have loaded sharedIndex

    const int idx = blockIdx.x * blockDim.y + threadIdx.y;

    if (idx < idxSize) { // Ensure the index is valid
        for (int startIdx = threadIdx.x + blockIdx.y * VT * WARP_SIZE; startIdx < stride; startIdx += VT * WARP_SIZE)
        {
            const long srcIdx = ((long) sharedIndex[idx] - 1) * stride;
            const long targetIdx = idx * stride;

            #pragma unroll
            for (int i = 0; i < VT; i++) {
                const int featureIdx = startIdx + i * WARP_SIZE;
                if (featureIdx < stride) {
                    tensor[targetIdx + featureIdx] = src[srcIdx + featureIdx];
                }
            }
        }
    }
}