#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* end = (unsigned int*) end_ptr;

    // Exit if block's starting pointer is beyond the end
    if (ptr >= end) return;

    // Use thread index to increase parallelization
    unsigned int idx = threadIdx.x;

    // Each thread writes its corresponding position
    for (unsigned int i = idx; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        ptr[i] = pattern;
    }
}