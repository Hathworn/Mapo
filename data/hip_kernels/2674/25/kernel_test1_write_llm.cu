#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    // Calculate the start pointer for this block
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Quick check if the current block's pointer is out of range
    if (ptr >= (unsigned long*)end_ptr) {
        return;
    }

    // Use threadIdx.x to iterate over elements within the block to maximize parallelism
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned long); i += blockDim.x) {
        ptr[i] = (unsigned long)&ptr[i];
    }
}