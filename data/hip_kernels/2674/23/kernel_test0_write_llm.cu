#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    unsigned int* orig_ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* ptr = orig_ptr;

    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);
    unsigned int pattern = 1;
    unsigned long mask = 4;

    *ptr = pattern;

    // Improved loop to avoid unnecessary casting
    while (mask < BLOCKSIZE) {
        ptr = (unsigned int*) (((unsigned long)orig_ptr) | mask); 

        if (ptr == orig_ptr || ptr >= block_end) {
            mask <<= 1;
            continue;
        }

        *ptr = pattern;
        pattern <<= 1;
        mask <<= 1;
    }
    return;
}