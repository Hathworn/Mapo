#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    // Optimize pointer arithmetic with shared memory for faster access
    extern __shared__ unsigned int shared_start_ptr[];

    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int* start_ptr = (unsigned int*) _start_ptr;

    // Copy start_ptr data into shared memory for faster reads
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        shared_start_ptr[i] = start_ptr[i];
    }
    __syncthreads();

    // Write from shared memory to global memory
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        ptr[i] = shared_start_ptr[i];
    }

    return;
}