#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    // Optimize pointer casting and initialization
    unsigned int* ptr = reinterpret_cast<unsigned int*>(_ptr);
    unsigned int* end_ptr = reinterpret_cast<unsigned int*>(_end_ptr);
    unsigned int* orig_ptr = ptr;
    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Initialize the first element
    *ptr = pattern;

    // Loop through while optimizing memory access
    while (ptr < end_ptr) {
        ptr = reinterpret_cast<unsigned int*>(reinterpret_cast<unsigned long>(orig_ptr) | mask);

        // Skip re-assignment and out-of-bounds condition checks
        if (ptr == orig_ptr) {
            mask <<= 1;
            continue;
        }
        if (ptr >= end_ptr) {
            break;
        }

        // Write pattern to the current pointer position
        *ptr = pattern;

        // Update pattern and mask
        pattern <<= 1;
        mask <<= 1;
    }
}