#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    // Calculate starting pointer for this block
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Exit if the pointer exceeds the end boundary
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Compute half of the required count of elements
    unsigned int half_count = BLOCKSIZE / sizeof(unsigned int) / 2;
    unsigned int* ptr_mid = ptr + half_count;

    // Loop unrolling for improved efficiency: Moving the first array to the second
    #pragma unroll
    for (unsigned int i = 0; i < half_count; i++) {
        ptr_mid[i] = ptr[i];
    }

    // Loop unrolling for moving back elements with an 8-element shift
    #pragma unroll
    for (unsigned int i = 0; i < half_count - 8; i++) {
        ptr[i + 8] = ptr_mid[i];
    }

    // Copy the last remaining 8 elements back to the start
    #pragma unroll
    for (unsigned int i = 0; i < 8; i++) {
        ptr[i] = ptr_mid[half_count - 8 + i];
    }

    return;
}