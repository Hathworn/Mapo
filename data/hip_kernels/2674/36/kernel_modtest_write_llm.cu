#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    unsigned int i;
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);

    // Check if pointer is within range to avoid unwanted memory access
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int limit = BLOCKSIZE / sizeof(unsigned int);

    // Combine two loops to improve performance and reduce global memory access
    for (i = 0; i < limit; i++) {
        if (i % MOD_SZ == offset) {
            ptr[i] = p1;
        } else {
            ptr[i] = p2;
        }
    }

    return;
}