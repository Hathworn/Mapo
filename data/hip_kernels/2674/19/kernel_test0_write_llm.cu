#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    // Calculate the starting pointer based on blockIdx only once
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Early return if out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Use a single statement to set all array elements to pattern
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        ptr[i] = pattern; // Pattern initialization
    }
}