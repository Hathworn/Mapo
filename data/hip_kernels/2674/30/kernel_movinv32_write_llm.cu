#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    // Calculate global thread index
    unsigned int global_tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int* ptr = (unsigned int*)(_ptr + global_tid * sizeof(unsigned int)); // Pointer arithmetic for the specific thread

    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Calculate the appropriate pattern shift and loop unrolling
    unsigned int k = offset + threadIdx.x * (BLOCKSIZE / blockDim.x) / sizeof(unsigned int);
    unsigned pat = pattern;
    unsigned int i;
    unsigned int stride = BLOCKSIZE / (sizeof(unsigned int) * blockDim.x); // Reduced calculations with stride

    for (i = 0; i < BLOCKSIZE / sizeof(unsigned int); i += stride) {
        // Unroll the loop to improve performance
        #pragma unroll
        for (unsigned int j = 0; j < stride; ++j) {
            ptr[i + j] = pat;
            k++;
            if (k >= 32) {
                k = 0;
                pat = lb;
            } else {
                pat = (pat << 1) | sval;
            }
        }
    }

    return;
}