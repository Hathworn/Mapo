#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate global thread ID
    const uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    const uint idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Calculate the number of threads in x and y dimensions
    const uint numThreadsX = blockDim.x * gridDim.x;
    const uint numThreadsY = blockDim.y * gridDim.y;
    
    // Optimize looping strategy by considering both x and y dimensions
    for (uint y = idy; y < tgtHeight; y += numThreadsY) {
        for (uint x = idx; x < tgtWidth; x += numThreadsX) {
            // Calculate source coordinates using modulo for tiling
            const uint srcY = y % srcHeight;
            const uint srcX = x % srcWidth;
            // Assign the tile-matched element from source to target
            tgt[y * tgtWidth + x] = src[srcY * srcWidth + srcX];
        }
    }
}