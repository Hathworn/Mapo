#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    // Initialize temporary variable for partial sum
    float tmp = 0.0f;

    // Calculate dot product using multiple threads and sum results
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Use one thread to sum up partial results across threads
    if (idx == 0) {
        for (int i = 1; i < blockDim.x; ++i) {
            reduction_buffer[0] += reduction_buffer[i];
        }
    }
    __syncthreads();

    // Compute gradient using the result from reduction
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}