#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define DP_BLOCKSIZE 512

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    // Initialize shared memory with 0
    shmem[threadIdx.x] = 0;
    // Ensure bounds checking for iteration over elements
    if (eidx < numElements) {
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            shmem[threadIdx.x] += a[eidx] * b[eidx];
        }
    }
    __syncthreads();

    // Perform parallel reduction in shared memory
    for (int stride = DP_BLOCKSIZE / 2; stride > 32; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }
    
    // Handle final warp-level reduction manually with volatile
    if (threadIdx.x < 32) {
        volatile float* mysh = &shmem[threadIdx.x];
        *mysh += mysh[32];
        *mysh += mysh[16];
        *mysh += mysh[8];
        *mysh += mysh[4];
        *mysh += mysh[2];
        *mysh += mysh[1];
        // Store result from the first thread in the block to target
        if (threadIdx.x == 0) {
            target[blockIdx.x] = *mysh;
        }
    }
}