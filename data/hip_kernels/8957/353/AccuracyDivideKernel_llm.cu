#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to optimize accuracy division
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use single thread to perform division for efficiency
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}