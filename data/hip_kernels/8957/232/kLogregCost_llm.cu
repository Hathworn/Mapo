#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Use a shared variable to store correct probability
        float correctProbability = 0;
        if (labelp == maxp) {
            int numMax = 0;

            // Use loop unrolling to reduce loop iterations
            for (int i = 0; i < numOut; i += 2) {
                numMax += (i < numOut && probs[i * numCases + tx] == maxp);
                if (i + 1 < numOut) {
                    numMax += (probs[(i + 1) * numCases + tx] == maxp);
                }
            }
            correctProbability = 1.0f / float(numMax);
        }
        correctProbs[tx] = correctProbability;
    }
}