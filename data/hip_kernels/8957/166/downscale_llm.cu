#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii /= d3;
    z = ii % d2;
    ii /= d2;
    y = ii % d1;
    ii /= d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii /= d3;
    z = ii % d2;
    ii /= d2;
    y = ii % d1;
    ii /= d1;
    x = ii;
    w /= scale_factor;
    z /= scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x;
    ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
    
    // Ensure the thread is within the bounds of the array
    if (ii >= no_elements) return;

    // Precompute commonly used values outside the loops
    int scaled_d2 = d2 * scale_factor;
    int scaled_d3 = d3 * scale_factor;

    // Loop unrolling for better performance
    for (int i = 0; i < scale_factor; i++) {
        for (int j = 0; j < scale_factor; j++) {
            int ipidx = (((ii / (d3 * d2)) * d1 + (ii % (d3 * d2)) / d3) * scaled_d2 + (ii % d3) * scale_factor + i) * scaled_d3 + j;
            gradInput_data[ii] += gradOutput_data[ipidx];
        }
    }
}