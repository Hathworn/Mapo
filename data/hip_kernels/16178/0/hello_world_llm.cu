#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_world(){
    // Each thread prints a message, but only one is needed here.
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Hello World\n");
    }
}