#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mykernel() {
    // Use thread synchronization to ensure ordered execution
    __syncthreads();
    
    // Check for a particular thread to output the message
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Hello world from device!\n");
    }
    
    __syncthreads();
} /* end kernel */