#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Use thread index for parallelism within block
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    c[idx] = a[idx] + b[idx];
}