#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using block and thread indexing
__global__ void add(int *a, int *b, int *c)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique index for each thread
    c[idx] = a[idx] + b[idx]; // Perform addition at each index
}