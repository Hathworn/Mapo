#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
    // Each thread computes one element
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}