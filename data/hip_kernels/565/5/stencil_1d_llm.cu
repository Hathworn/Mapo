#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil_1d(int n, double *in, double *out)
{
    // Allocate shared memory with padding for handling halos
    extern __shared__ double temp[];

    // Calculate global and local indices
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int localIndex = threadIdx.x + RADIUS;

    // Check boundary, exit if globalIndex is out of bounds
    if (globalIndex >= n) return;

    // Load main data into shared memory
    temp[localIndex] = in[globalIndex];

    // Load halo data into shared memory
    if (threadIdx.x < RADIUS) {
        if (globalIndex >= RADIUS) {
            temp[localIndex - RADIUS] = in[globalIndex - RADIUS];
        }
        if (globalIndex < n - RADIUS) {
            temp[localIndex + THREADS_PER_BLOCK] = in[globalIndex + THREADS_PER_BLOCK];
        }
    }

    __syncthreads();

    // Handle boundary conditions: output zero contribution for border
    if (globalIndex < RADIUS || globalIndex >= (n - RADIUS)) {
        out[globalIndex] = (double)globalIndex * ( (double)RADIUS*2 + 1);
        return;
    }

    // Compute stencil in shared memory
    double result = 0.0;
    for (int i = -RADIUS; i <= RADIUS; i++) {
        result += temp[localIndex + i];
    }

    // Store result in global memory
    out[globalIndex] = result;
}