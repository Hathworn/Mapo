#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil_1d(int n, double *in, double *out)
{
    // Allocate shared memory for the computation
    __shared__ double temp[THREADS_PER_BLOCK + 2*(RADIUS)];

    // Calculate global and local indices
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int localIndex = threadIdx.x + RADIUS;

    // Early return if the global index is out of bounds
    if (globalIndex >= n) return;

    // Load elements into shared memory
    temp[localIndex] = in[globalIndex];
    
    // Load halo elements with boundary checks to avoid overflows
    if (threadIdx.x < RADIUS) {
        if (globalIndex >= RADIUS) {
            temp[localIndex - RADIUS] = in[globalIndex - RADIUS];
        }
        if (globalIndex < (n - RADIUS) && globalIndex + THREADS_PER_BLOCK < n) {
            temp[localIndex + THREADS_PER_BLOCK] = in[globalIndex + THREADS_PER_BLOCK];
        }
    }

    // Handle boundary conditions explicitly
    if (globalIndex < RADIUS || globalIndex >= (n - RADIUS)) {
        out[globalIndex] = (double)globalIndex * ((double)RADIUS * 2 + 1);
        return;
    }

    // Compute stencil operation
    double result = 0.0;
    for (int i = -RADIUS; i <= RADIUS; i++) {
        result += temp[localIndex + i];
    }

    // Write the result to the output array
    out[globalIndex] = result;
}