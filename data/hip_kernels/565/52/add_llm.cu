#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Load inputs into registers for faster access
    int reg_a = a[index];
    int reg_b = b[index];
    
    // Perform addition using registers and store the result in global memory
    c[index] = reg_a + reg_b;
}