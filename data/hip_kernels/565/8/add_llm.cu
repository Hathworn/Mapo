#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel launch by ensuring warp size processing
__global__ void add(int *a, int *b, int *c)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique global index for the thread
    c[idx] = a[idx] + b[idx]; // Perform the addition using global index
}