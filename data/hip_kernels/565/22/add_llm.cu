#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to optimize memory access (assuming inputs are smaller than shared memory size)
    __shared__ int shared_a[256];  // Adjust size to match blockDim.x
    __shared__ int shared_b[256];

    // Load data into shared memory
    shared_a[threadIdx.x] = a[index];
    shared_b[threadIdx.x] = b[index];
    __syncthreads();

    // Perform addition using shared memory
    c[index] = shared_a[threadIdx.x] + shared_b[threadIdx.x];
}