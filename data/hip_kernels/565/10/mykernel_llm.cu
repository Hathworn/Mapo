#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mykernel() {
    // Use thread and block indices to ensure unique output per thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello world from device! Thread id: %d\n", idx);
} /* end kernel */