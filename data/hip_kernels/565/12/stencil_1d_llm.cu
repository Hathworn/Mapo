#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil_1d(int n, double *in, double *out)
{
    // Calculate global index in the array
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Return if my global index is larger than the array size
    if (globalIndex >= n) return;

    // Handle the boundary conditions
    if (globalIndex < RADIUS || globalIndex >= (n - RADIUS)) {
        out[globalIndex] = (double)globalIndex * ((double)RADIUS * 2 + 1);
        return;
    }

    double result = 0.0;

    // Unroll loop for better performance
    #pragma unroll
    for (int i = globalIndex - RADIUS; i <= globalIndex + RADIUS; i++) {
        result += in[i];
    }

    out[globalIndex] = result;
}