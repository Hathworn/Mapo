#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

// Optimized Kernel function
__global__ void Kernel_polimoniale(float *Vd, float *Ris, int N, int C, int dim_indici, int *ind, int *Vp, int *Vnp, int nr_max_val, int s)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Cache y in a register for reduced global memory access
    int local_y = y;

    while (x < N) {
        while (local_y < dim_indici) {
            
            // Load tmp_ind into registers to reduce global memory access
            int tmp_ind = ind[local_y];
            float tmp = 1.0; // Use registers for tmp
            float pol = 0.0; // Use registers for pol

            int Nr_val = Vnp[x];

            // Unroll loop for better instruction-level parallelism
            #pragma unroll
            for (int j = 0; j < Nr_val; j++) {
                int pos = Vp[x * nr_max_val + j];
                pol += Vd[x * C + pos] * Vd[tmp_ind * C + pos];
            }

            pol += 1;

            // Use multiplication instead of a loop for power calculations
            float pol_power = pol;
            for (int j = 1; j < s; j++) {
                pol_power *= pol;
            }

            Ris[local_y * N + x] = pol_power;
            local_y += blockDim.y * gridDim.y;
        }
        x += blockDim.x * gridDim.x;
    }
}