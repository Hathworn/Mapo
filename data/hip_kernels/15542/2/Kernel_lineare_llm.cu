#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized Kernel function
__global__ void Kernel_lineare(float *Vd, float *Ris, int N, int C, int dim_indici, int *ind, int *Vp, int *Vnp, int nr_max_val)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int stride_x = blockDim.x * gridDim.x; // Loop stride for x

    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int stride_y = blockDim.y * gridDim.y; // Loop stride for y

    while (x < N) 
    {
        while (y < dim_indici)
        {
            int tmp_ind = ind[y];
            float lin = 0.0f;
            int Nr_val = Vnp[x];

            for(int j = 0; j < Nr_val; j++)
            {
                int pos = Vp[x * nr_max_val + j];
                lin += Vd[x * C + pos] * Vd[tmp_ind * C + pos];
            }
            
            Ris[y * N + x] = lin;
            y += stride_y; // Increment y by its stride
        }
        x += stride_x; // Increment x by its stride
    }
}