#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimized for better performance
__global__ void Kernel_norme(float *Vd, float *Nd, int *Vp, int *Vnp, int N, int C, int nr_max_val)
{
    // Calculate global thread index
    long int x = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure the thread operates only within valid range
    if(x < N)
    {
        float norma = 0;  // Initialize norma to zero

        // Fetch number of relevant values from shared memory for efficiency
        int Nr_val = Vnp[x];

        // Loop over the relevant values for computation
        for(int i = 0; i < Nr_val; i++)
        {
            int pos = Vp[x * nr_max_val + i];  // Get position from Vp
            float val = Vd[x * C + pos];  // Fetch value from Vd for computation
            norma += val * val;  // Accumulate square of value to norma
        }

        // Write norma result back to global memory
        Nd[x] = norma;
    }
}