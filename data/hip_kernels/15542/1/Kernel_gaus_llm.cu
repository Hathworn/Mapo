#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void Kernel_gaus(float *Vd, float *Ris, float *Nd, int N, int C, int dim_indici, int *ind, float sigma, int *Vp, int *Vnp, int nr_max_val)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= N || y >= dim_indici) return; // Boundary check

    int j;
    int pos;
    int tmp_ind = ind[y];
    float gaus = 0.0;

    int Nr_val = Vnp[x];

    for(j = 0; j < Nr_val; j++)
    {
        pos = Vp[x * nr_max_val + j];
        gaus += (Vd[x * C + pos] * Vd[tmp_ind * C + pos]); // Use of compound assignment
    }

    gaus = -2.0f * gaus + Nd[x] + Nd[tmp_ind];
    gaus = exp(-gaus * sigma); // Move operation to a separate line for clarity

    Ris[y * N + x] = gaus;
}