#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindClosestPoint(float3 *points, int *closestPoint, const int numberPoints)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check to ensure valid index
    if (idx >= numberPoints) return;

    // Initialize minimal distance with a large number
    int distanceBetweenPoints = INT_MAX;
    int tempDistance;
    
    // Iterate over all points to find the closest one
    for (int j = 0; j < numberPoints; j++)
    {
        // Skip calculating distance from point to itself
        if (idx != j)
        {
            // Calculate squared distance to avoid sqrt and improve performance
            tempDistance = (points[idx].x - points[j].x) * (points[idx].x - points[j].x)
                         + (points[idx].y - points[j].y) * (points[idx].y - points[j].y);

            // Update the closest point if a closer one is found
            if (tempDistance < distanceBetweenPoints)
            {
                distanceBetweenPoints = tempDistance;
                closestPoint[idx] = j;
            }
        }
    }
}