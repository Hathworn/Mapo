#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AddIntegers(int *a, int *b)
{
    // Use thread index to allow for potential parallel execution
    int idx = threadIdx.x;
    if(idx == 0) {
        a[0] += b[0];
    }
}