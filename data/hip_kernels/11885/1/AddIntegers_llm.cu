#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AddIntegers(int *arr1, int *arr2, int num_elements)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a loop to process more than one element per thread if necessary
    for (int i = id; i < num_elements; i += blockDim.x * gridDim.x)
    {
        arr1[i] += arr2[i];
    }
}