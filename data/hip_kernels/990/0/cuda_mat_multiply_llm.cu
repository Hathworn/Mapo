#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_mat_multiply(const double* A, const double* B, double* C, int rowsa, int colsa, int rowsb, int colsb, int rowsc, int colsc) {
    __shared__ double sA[32][32];   // Tile size of 32x32
    __shared__ double sB[32][32];
    int Row = blockDim.y * blockIdx.y + threadIdx.y;
    int Col = blockDim.x * blockIdx.x + threadIdx.x;
    double Cvalue = 0.0;

    for (int k = 0; k < (((colsa - 1) / 32) + 1); k++) {
        // Load A into shared memory if within bounds
        if ((Row < rowsa) && (threadIdx.x + (k * 32)) < colsa) {
            sA[threadIdx.y][threadIdx.x] = A[Row * colsa + threadIdx.x + (k * 32)];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }

        // Load B into shared memory if within bounds
        if (Col < colsb && (threadIdx.y + k * 32) < rowsb) {
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k * 32) * colsb + Col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads(); // Ensure all threads have loaded data

        // Perform computation within shared memory tiles
        for (int j = 0; j < 32; ++j) {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }

        __syncthreads(); // Ensure all threads have completed computation
    }

    // Store result in C if within bounds
    if (Row < rowsc && Col < colsc) {
        C[Row * colsc + Col] = Cvalue;
    }
}