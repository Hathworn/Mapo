#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_mat_transpose(const double* __restrict__ src, double* __restrict__ dst, int colssrc, int colsdst, int n) {
    // Calculate unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;

    // Loop handling using stride
    for (int idx = tid; idx < n; idx += total_threads) {
        // Calculate row and column indices for source and destination
        int cdst = idx % colsdst;
        int rdst = idx / colsdst;
        int rsrc = cdst;
        int csrc = rdst;
        
        // Perform matrix transpose
        dst[idx] = src[rsrc * colssrc + csrc];
    }
}