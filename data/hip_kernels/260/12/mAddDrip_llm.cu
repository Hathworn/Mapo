#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mAddDrip(float *dense, int centerX, int centerY, float radius) {
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate the thread's global position directly
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate squared distance to avoid using sqrt
    float distSquared = (x - centerX) * (x - centerX) + (y - centerY) * (y - centerY);

    // Compare squared distance with squared radius
    if (distSquared < radius * radius) {
        dense[Idx] += 200.0f;
    }
}