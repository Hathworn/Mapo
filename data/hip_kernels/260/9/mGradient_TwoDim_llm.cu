#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ bool checkBoundary(int blockIdx, int blockDim, int threadIdx) {
    int x = threadIdx;
    int y = blockIdx;
    return (x == 0 || x == (blockDim - 1) || y == 0 || y == 479);
}

__global__ void mGradient_TwoDim(float *u_dimX, float *u_dimY, float *scalar, float coeffX, float coeffY) {
    // Calculate the unique thread index
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;
    int Left, Right, Top, Bottom;

    // Quick return if on the boundary to avoid warp divergence
    if(checkBoundary(blockIdx.x, blockDim.x, threadIdx.x)) return;

    // Precompute indices for better memory access
    Left = Idx - 1;
    Right = Idx + 1;
    Top = Idx + blockDim.x;
    Bottom = Idx - blockDim.x;

    // Calculate the gradients using coeffs and neighboring values
    u_dimX[Idx] -= (scalar[Right] - scalar[Left]) * coeffX;
    u_dimY[Idx] -= (scalar[Top] - scalar[Bottom]) * coeffY;
}