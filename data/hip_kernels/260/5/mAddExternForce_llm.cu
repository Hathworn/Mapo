#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mAddExternForce(float *w_dimX, float *w_dimY, float *f_dimX, float *f_dimY, float dt) {
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (Idx < gridDim.x * blockDim.x) { // Ensure Idx is within bounds
        float halfFactor = -0.5f; // Define a constant factor
        w_dimX[Idx] *= halfFactor;
        w_dimY[Idx] *= halfFactor; // Use compound assignment
    }
}