#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mAttachTexture(uint8_t *frame, float *dense) {
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;
    float value = dense[Idx]; // Cache dense value to avoid repeated memory accesses
    frame[Idx] = (value > 255.0f) ? 255 : static_cast<uint8_t>(value); // Use static_cast for clarity
}