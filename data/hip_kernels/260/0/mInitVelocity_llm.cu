#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mInitVelocity(float *u_dimX, float *u_dimY) {
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Prevent out-of-bounds access
    if (Idx < gridDim.x * blockDim.x) {
        u_dimX[Idx] = 0.f;

        // Cache the reciprocal of (blockIdx.x + 1) to avoid redundant computation
        float reciprocal = 0.8f / (float)(blockIdx.x + 1);
        u_dimY[Idx] = reciprocal;
    }
}