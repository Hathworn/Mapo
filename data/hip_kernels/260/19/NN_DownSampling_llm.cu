#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NN_DownSampling(float *target, const float *source, const int wt, const int ht, const int ws, const int hs)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Move condition inside to reduce register pressure
    if (y < ht && x < wt) {
        int curt = y * wt + x;
        int curs = (y * 2) * ws + (x * 2);
        
        // Unroll loop to reduce overhead
        #pragma unroll
        for (int i = 0; i < 3; ++i) {
            target[curt * 3 + i] = source[curs * 3 + i];
        }
    }
}