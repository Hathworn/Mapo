#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mInitForce(float *f_dimX, float *f_dimY) {
    // Calculate unique thread index
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Pre-calculate constants for center point
    const float centerX = 320.0f;
    const float centerY = 240.0f;

    // Calculate thread's x and y coordinates
    float x = (float)threadIdx.x;
    float y = (float)blockIdx.x;

    // Compute length only once
    float dx = x - centerX;
    float dy = y - centerY;
    float length = sqrt(dx * dx + dy * dy);

    // Set force values based on the distance from center
    if(length < SWIRL_RADIUS) {
        f_dimX[Idx] = dy / length;
        f_dimY[Idx] = dx / length;
    } else {
        f_dimX[Idx] = f_dimY[Idx] = 0.f;
    }
}