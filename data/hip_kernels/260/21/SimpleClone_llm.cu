#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SimpleClone(const float *background, const float *target, const float *mask, float *output, const int wb, const int hb, const int wt, const int ht, const int oy, const int ox)
{
    // Calculate current thread position within target
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    if (yt >= ht || xt >= wt) return; // Early exit for threads outside target

    const int curt = wt * yt + xt;

    if (mask[curt] <= 127.0f) return; // Early exit if mask is not significant

    // Calculate position in the background
    const int yb = oy + yt, xb = ox + xt;
    if (yb < 0 || yb >= hb || xb < 0 || xb >= wb) return; // Early exit if outside background

    // Calculate absolute position in output
    const int curb = wb * yb + xb;

    // Copy RGB values from target to output
    #pragma unroll // Unroll loop for better performance
    for (int i = 0; i < 3; ++i) {
        output[curb * 3 + i] = target[curt * 3 + i];
    }
}