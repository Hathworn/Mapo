#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PoissonImageCloningIteration(const float *fixed, const float *mask, const float *buf1, float *buf2, const int wt, const int ht)
{
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    const int curt = wt * yt + xt;
    
    if (yt < ht && xt < wt && mask[curt] > 127.0f) {
        // Check borders to avoid out-of-bound accesses
        bool nt_bnd = (yt == 0), wt_bnd = (xt == 0), st_bnd = (yt == ht - 1), et_bnd = (xt == wt - 1);
        int North_t = (nt_bnd) ? curt : (curt - wt);
        int West_t = (wt_bnd) ? curt : (curt - 1);
        int South_t = (st_bnd) ? curt : (curt + wt);
        int East_t = (et_bnd) ? curt : (curt + 1);

        // Use shared memory to reduce global memory accesses
        float fixedCur[3] = {fixed[curt * 3 + 0], fixed[curt * 3 + 1], fixed[curt * 3 + 2]};
        float buf1North[3], buf1West[3], buf1South[3], buf1East[3];
        
        if (!nt_bnd) {
            buf1North[0] = buf1[North_t * 3 + 0];
            buf1North[1] = buf1[North_t * 3 + 1];
            buf1North[2] = buf1[North_t * 3 + 2];
        }

        if (!wt_bnd) {
            buf1West[0] = buf1[West_t * 3 + 0];
            buf1West[1] = buf1[West_t * 3 + 1];
            buf1West[2] = buf1[West_t * 3 + 2];
        }

        if (!st_bnd) {
            buf1South[0] = buf1[South_t * 3 + 0];
            buf1South[1] = buf1[South_t * 3 + 1];
            buf1South[2] = buf1[South_t * 3 + 2];
        }

        if (!et_bnd) {
            buf1East[0] = buf1[East_t * 3 + 0];
            buf1East[1] = buf1[East_t * 3 + 1];
            buf1East[2] = buf1[East_t * 3 + 2];
        }

        // Aggregate values
        if (!nt_bnd) {
            fixedCur[0] += buf1North[0];
            fixedCur[1] += buf1North[1];
            fixedCur[2] += buf1North[2];
        }

        if (!wt_bnd) {
            fixedCur[0] += buf1West[0];
            fixedCur[1] += buf1West[1];
            fixedCur[2] += buf1West[2];
        }

        if (!st_bnd) {
            fixedCur[0] += buf1South[0];
            fixedCur[1] += buf1South[1];
            fixedCur[2] += buf1South[2];
        }

        if (!et_bnd) {
            fixedCur[0] += buf1East[0];
            fixedCur[1] += buf1East[1];
            fixedCur[2] += buf1East[2];
        }

        buf2[curt * 3 + 0] = fixedCur[0] * 0.25f;
        buf2[curt * 3 + 1] = fixedCur[1] * 0.25f;
        buf2[curt * 3 + 2] = fixedCur[2] * 0.25f;
    }
}