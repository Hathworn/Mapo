#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalculateFixed( const float *background, const float *target, const float *mask, float *fixed, const int wb, const int hb, const int wt, const int ht, const int oy, const int ox )
{
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    const int curt = wt * yt + xt;
    
    if (yt < ht && xt < wt && mask[curt] > 127.0f) {
        // Boundary flags
        bool nt_bnd = (yt == 0), wt_bnd = (xt == 0), st_bnd = (yt == ht - 1), et_bnd = (xt == wt - 1);
        
        // Offset indices
        int North_t = (nt_bnd) ? curt : (curt - wt);
        int West_t  = (wt_bnd) ? curt : (curt - 1);
        int South_t = (st_bnd) ? curt : (curt + wt);
        int East_t  = (et_bnd) ? curt : (curt + 1);

        // Precompute repeated target index access
        const float* target_cur = target + curt * 3;
        const float* target_N   = target + North_t * 3;
        const float* target_W   = target + West_t * 3;
        const float* target_S   = target + South_t * 3;
        const float* target_E   = target + East_t * 3;

        // Compute fixed values
        float* fixed_cur = fixed + curt * 3;
        for (int i = 0; i < 3; ++i) {
            fixed_cur[i] = 4.0f * target_cur[i] - (target_N[i] + target_W[i] + target_S[i] + target_E[i]);
        }

        const int yb = oy + yt, xb = ox + xt;
        const int curb = wb * yb + xb;
        
        if (0 <= yb && yb < hb && 0 <= xb && xb < wb) {
            // Background boundary flags
            bool nb_bnd = (yb == 0), wb_bnd = (xb == 0), sb_bnd = (yb == hb - 1), eb_bnd = (xb == wb - 1);
            
            // Offset indices for background
            int North_b = (nb_bnd) ? curb : (curb - wb);
            int West_b  = (wb_bnd) ? curb : (curb - 1);
            int South_b = (sb_bnd) ? curb : (curb + wb);
            int East_b  = (eb_bnd) ? curb : (curb + 1);

            // Mask checks
            bool isMasked_n = (nt_bnd) ? true : (mask[North_t] <= 127.0f);
            bool isMasked_w = (wt_bnd) ? true : (mask[West_t] <= 127.0f);
            bool isMasked_s = (st_bnd) ? true : (mask[South_t] <= 127.0f);
            bool isMasked_e = (et_bnd) ? true : (mask[East_t] <= 127.0f);

            // Apply background additions
            float* background_N = background + North_b * 3;
            float* background_W = background + West_b * 3;
            float* background_S = background + South_b * 3;
            float* background_E = background + East_b * 3;

            if (isMasked_n) {
                for (int i = 0; i < 3; ++i) {
                    fixed_cur[i] += background_N[i];
                }
            }
            if (isMasked_w) {
                for (int i = 0; i < 3; ++i) {
                    fixed_cur[i] += background_W[i];
                }
            }
            if (isMasked_s) {
                for (int i = 0; i < 3; ++i) {
                    fixed_cur[i] += background_S[i];
                }
            }
            if (isMasked_e) {
                for (int i = 0; i < 3; ++i) {
                    fixed_cur[i] += background_E[i];
                }
            }
        }
    }
}