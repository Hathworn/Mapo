#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ bool checkBoundary(int x, int y, int blockDim_x, int gridDim_y) {
    return (x == 0 || x == (blockDim_x-1) || y == 0 || y == (gridDim_y-1));
}

__global__ void mSetFieldBoundary(float *field, float scalar) {
    int x = threadIdx.x;
    int y = blockIdx.x;
    if (checkBoundary(x, y, blockDim.x, gridDim.x)) {
        int Idx = y * blockDim.x + x;
        
        // Simplify boundary condition checks and field updates
        if (x == 0) {
            if (y == 0) {
                field[Idx] = field[Idx + blockDim.x + 1] * scalar;
            } else if (y == gridDim.x - 1) {
                field[Idx] = field[Idx - blockDim.x + 1] * scalar;
            } else {
                field[Idx] = field[Idx + 1] * scalar;
            }
        } else if (x == blockDim.x - 1) {
            if (y == 0) {
                field[Idx] = field[Idx + blockDim.x - 1] * scalar;
            } else if (y == gridDim.x - 1) {
                field[Idx] = field[Idx - blockDim.x - 1] * scalar;
            } else {
                field[Idx] = field[Idx - 1] * scalar;
            }
        } else {
            if (y == 0) {
                field[Idx] = field[Idx + blockDim.x] * scalar;
            } else {
                field[Idx] = field[Idx - blockDim.x] * scalar;
            }
        }
    }
}