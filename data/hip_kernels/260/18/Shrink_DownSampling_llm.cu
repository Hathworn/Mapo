#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Shrink_DownSampling(float *target, const float *source, const int wt, const int ht, const int ws, const int hs)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if(y < ht && x < wt) {
        const int curt = (y * wt + x) * 3; // Optimize memory access pattern
        const int curs = (y * 2 * ws + x * 2) * 3;
        
        float sumR = 0.0f, sumG = 0.0f, sumB = 0.0f;
        
        // Unroll loops and use common subexpression elimination for source indexing
        for(int dy = 0; dy < 2; ++dy) {
            for(int dx = 0; dx < 2; ++dx) {
                int idx = curs + (dy * ws + dx) * 3;
                sumR += source[idx + 0];
                sumG += source[idx + 1];
                sumB += source[idx + 2];
            }
        }
        
        target[curt + 0] = sumR * 0.25f;
        target[curt + 1] = sumG * 0.25f;
        target[curt + 2] = sumB * 0.25f;
    }
}