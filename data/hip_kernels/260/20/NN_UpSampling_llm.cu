#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NN_UpSampling(float *target, const float *source, const int wt, const int ht)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (y < ht && x < wt) {
        int curt = y * wt + x;
        int curs = (y / 2) * (wt / 2) + x / 2;
        // Using registers for temporary storage to minimize memory accesses
        float source_r = source[curs * 3 + 0];
        float source_g = source[curs * 3 + 1];
        float source_b = source[curs * 3 + 2];
        target[curt * 3 + 0] = source_r;
        target[curt * 3 + 1] = source_g;
        target[curt * 3 + 2] = source_b;
    }
}