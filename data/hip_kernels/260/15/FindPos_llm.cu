#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindPos(int *pos, bool *forest, int text_size, int order, int step)
{
    // Use shared memory to reduce global memory access
    extern __shared__ bool shared_forest[];
    int text_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_local_idx = threadIdx.x + blockDim.x; // Local index for current block
    shared_forest[threadIdx.x] = forest[blockIdx.x * blockDim.x + threadIdx.x];
    __syncthreads();

    if (text_idx < text_size) {
        if (!shared_forest[threadIdx.x + blockDim.x]) {
            pos[text_idx] = 0;
        } else {
            bool isCurBlock = true;
            bool isLeftMost = (blockIdx.x < 1);
            int nodeIdx = thread_local_idx;
            int leftBound = blockDim.x;
            int rightBound = 2 * blockDim.x - 1;
            int alignOrder = 0;

            // bottom-up
            while (alignOrder != order) {
                int leftInx;
                if (nodeIdx - 1 < leftBound) {
                    if (isLeftMost) break;
                    isCurBlock = false;
                    leftInx = offset - step + rightBound;
                } else {
                    leftInx = nodeIdx - 1;
                }

                if (!forest[blockIdx.x * step + leftInx]) break;

                rightBound = leftBound - 1;
                leftBound /= 2;
                nodeIdx /= 2;
                alignOrder++;
            }

            // top-down
            if (alignOrder == order && !isLeftMost) isCurBlock = false;
            nodeIdx = (!isCurBlock) ? rightBound
                    : (nodeIdx - 1 < leftBound) ? nodeIdx
                    : nodeIdx - 1;

            while (alignOrder != 0) {
                if ((alignOrder == order && isCurBlock) || forest[blockIdx.x * step + 2 * nodeIdx + 1]) {
                    nodeIdx = 2 * nodeIdx;
                } else {
                    nodeIdx = 2 * nodeIdx + 1;
                }
                alignOrder--;
            }

            pos[text_idx] = (isCurBlock) ? (threadIdx.x - (nodeIdx - blockDim.x) + (forest[blockIdx.x * step + nodeIdx]))
                           : (step - nodeIdx + threadIdx.x);
        }
    }
}