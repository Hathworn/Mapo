#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void IndexLeafNode(const char *text, bool *forest, int text_size, int step)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the position in forest array using idx instead of offset
    if (idx < text_size) // Check idx bound first to avoid unnecessary work
    {
        forest[idx] = (text[idx] != '\n');
    }
}