#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ bool checkBoundary(int blockIdx, int blockDim, int threadIdx) {
    int x = threadIdx;
    int y = blockIdx;
    return (x == 0 || x == (blockDim-1) || y == 0 || y == 479);
}

__global__ void mDivergence_TwoDim(float *div, float *u_dimX, float *u_dimY, float r_sStep) {
    // Calculate the global thread id
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if the current thread is at the boundary, and return early if true
    if (checkBoundary(blockIdx.x, blockDim.x, threadIdx.x)) return;
    
    // Calculate neighbors using precomputed Idx
    int Left   = Idx - 1;
    int Right  = Idx + 1;
    int Top    = Idx + blockDim.x;
    int Bottom = Idx - blockDim.x;
    
    // Calculate divergence and store in output
    div[Idx] = ((u_dimX[Right]-u_dimX[Left])+(u_dimY[Top]-u_dimY[Bottom]))*r_sStep;
}