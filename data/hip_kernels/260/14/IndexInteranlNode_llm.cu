#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void IndexInternalNode(bool *forest, int base, int step)
{
    // Calculate the offset based on block and thread indices once
    int offset = blockIdx.x * step;
    int left  = 2 * (base + threadIdx.x) + offset;
    int right = left + 1;

    // Move offset addition out of individual calculations
    forest[left / 2] = forest[left] && forest[right];
}