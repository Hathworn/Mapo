#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ bool checkBoundary(int blockIdx, int blockDim, int threadIdx) {
    int x = threadIdx;
    int y = blockIdx;
    return (x == 0 || x == (blockDim - 1) || y == 0 || y == 479);
}

__global__ void mAdvect(float *new_data, float *old_data, float *xv, float *yv, float t_step, float s_stepX, float s_stepY) {
    // Load thread and block indices
    int x = threadIdx.x;
    int y = blockIdx.x;

    // Early exit for boundary conditions
    if (checkBoundary(y, blockDim.x, x)) return;

    // Compute unique linear index
    int Idx = y * blockDim.x + x;

    // Calculate positions 
    float curr_x = static_cast<float>(x);
    float curr_y = static_cast<float>(y);
    float last_x = curr_x - t_step * s_stepX * xv[Idx];
    float last_y = curr_y - t_step * s_stepY * yv[Idx];

    // Clamp positions to avoid out-of-bound access
    last_x = fminf(fmaxf(last_x, 1.5f), 637.5f);
    last_y = fminf(fmaxf(last_y, 1.5f), 477.5f);

    // Bilinear interpolation calculations
    float xDiff = last_x - floorf(last_x);
    float yDiff = last_y - floorf(last_y);
    int LeftTopX = static_cast<int>(last_x);
    int LeftTopY = static_cast<int>(last_y);
    int base = LeftTopY * blockDim.x + LeftTopX; // Pre-calculate base index

    // Efficient memory access with reuse of base index
    new_data[Idx] = (xDiff * yDiff) * old_data[base + blockDim.x + 1]
                  + (xDiff * (1.f - yDiff)) * old_data[base + 1]
                  + ((1.f - xDiff) * yDiff) * old_data[base + blockDim.x]
                  + ((1.f - xDiff) * (1.f - yDiff)) * old_data[base];
}