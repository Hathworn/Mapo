#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mAddDensity(float *dense, float *dense_old, float dt) {
    // Use shared memory for faster access than global memory
    __shared__ float shared_dense_old[256]; // assuming blockDim.x <= 256
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;
    shared_dense_old[threadIdx.x] = dense_old[Idx];
    __syncthreads(); // Ensure all threads have loaded their data

    // Perform computation after ensuring data is available
    dense[Idx] += shared_dense_old[threadIdx.x] * dt;
}