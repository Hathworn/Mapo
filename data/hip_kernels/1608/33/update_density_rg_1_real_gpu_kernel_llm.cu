#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_density_rg_1_real_gpu_kernel(int size__, double const* __restrict__ psi_rg__, double wt__, double* __restrict__ density_rg__)
{
    int ir = blockIdx.x * blockDim.x + threadIdx.x;
    if (ir < size__) {
        double p = psi_rg__[ir];
        density_rg__[ir] += p * p * wt__; // Utilize restrict to hint compiler for potential optimizations
    }
}