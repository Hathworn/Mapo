#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_by_veff_real_real_gpu_kernel(int nr__, double* buf__, double const* veff__)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop with step size equal to total number of threads
    for (; i < nr__; i += blockDim.x * gridDim.x) {
        buf__[i] *= veff__[i];
    }
}