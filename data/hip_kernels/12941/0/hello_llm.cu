#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel by improving memory access pattern
__global__ void hello(char *a, int *b) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    a[idx] += b[idx]; // Use global index to access memory more efficiently
}