#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitonic_sort_step(unsigned int *dev_values, int j, int k) {
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ixj = i ^ j;

    // Check if the threads have valid sorting partners
    if (ixj > i) {
        // Determine whether to sort ascending or descending based on bitwise comparison
        bool shouldSortAscending = (i & k) == 0;

        // Use a ternary operation to reduce redundant code
        unsigned int temp = dev_values[i];
        bool condition = (dev_values[i] > dev_values[ixj]);
        
        // Use conditional statements to perform sorting
        if (shouldSortAscending && condition) {
            dev_values[i] = dev_values[ixj];
            dev_values[ixj] = temp;
        } else if (!shouldSortAscending && !condition) {
            dev_values[i] = dev_values[ixj];
            dev_values[ixj] = temp;
        }
    }
}