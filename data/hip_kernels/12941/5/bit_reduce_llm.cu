#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bit_reduce(const uint32_t *input_array, uint32_t *intBuf) {
    uint32_t i = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t a = input_array[i];

    // Precompute the address offset for copying to intBuf to minimize repeated computations
    uint8_t *bufOffset = ((uint8_t *)(&intBuf[i]) + 1);

    if (a <= 0xff) {
        intBuf[i] = 1;  // Store size directly
        *(uint8_t *)bufOffset = static_cast<uint8_t>(a);  // Use direct assignment for better performance
    } else if (a <= 0xffff) {
        intBuf[i] = 2;  // Store size directly
        *(uint16_t *)bufOffset = static_cast<uint16_t>(a);  // Use direct assignment for better performance
    } else {
        intBuf[i] = 4;  // Store size directly
        *(uint32_t *)bufOffset = a;  // Use direct assignment for better performance
    }
}