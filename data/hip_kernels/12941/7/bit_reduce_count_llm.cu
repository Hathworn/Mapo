#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bit_reduce_count(const uint32_t *input_array, uint32_t *intBuf, uint32_t *countBuf, const uint16_t dataCount) {
    extern __shared__ uint32_t sharedMem[];

    const uint32_t i = threadIdx.x + blockDim.x * blockIdx.x;
    const uint32_t a = (i < dataCount) ? input_array[i] : 0; // Ensure within bounds
    uint8_t size = 0;

    // Determine the size using a streamlined conditional
    size = (a <= 0xff) ? 1 : (a <= 0xffff) ? sizeof(uint16_t) : sizeof(uint32_t);

    sharedMem[threadIdx.x] = size;
    __syncthreads();

    // Use one thread to compute the prefix sums in shared memory
    if (threadIdx.x == 0) {
        uint32_t total = 0;
        for (uint16_t j = 0; j < blockDim.x && j < dataCount - blockDim.x * blockIdx.x; j++) { // Boundary check
            total += sharedMem[j];
            sharedMem[j] = total;
        }
        countBuf[blockIdx.x] = total;
    }
    __syncthreads();

    // Simplify writeindex calculation and ensure alignment
    uint8_t* writeindex = (threadIdx.x > 0 ? sharedMem[threadIdx.x - 1] : 0) + ((uint8_t*)&intBuf[blockDim.x * blockIdx.x]);
    
    // Reduce branching by setting the value and using one memcpy call
    *writeindex = size;
    memcpy(writeindex+1, &a, size); // Use the determined size
}