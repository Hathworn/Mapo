#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void seq_compact(uint8_t *intBuf, const uint16_t dataSize, uint32_t *sizeBuf) {
    // Use shared memory to reduce global memory access latency
    __shared__ uint8_t sharedIntBuf[1024];
    
    uint16_t writeIndex = 0;
    
    for (uint16_t i = threadIdx.x; i < dataSize; i += blockDim.x) {
        // Use coalesced memory access to improve performance
        const uint16_t readIndex = i * 4;
        uint8_t size = intBuf[readIndex];
        
        // Copy to shared memory first to avoid repeated global memory access
        for (uint8_t j = 0; j <= size; ++j) {
            sharedIntBuf[writeIndex + j] = intBuf[readIndex + j];
        }
        
        writeIndex += size + 1;
    }

    // Copy data back to global memory
    for (uint16_t i = threadIdx.x; i < writeIndex; i += blockDim.x) {
        intBuf[i] = sharedIntBuf[i];
    }

    // Update sizeBuf with the final write index
    if (threadIdx.x == 0) {
        atomicMax(&sizeBuf[0], writeIndex);
    }

    // Zero out the rest of the buffer
    const uint32_t int_buf_size = (dataSize * sizeof(uint32_t)) + (dataSize * sizeof(uint8_t));
    if (threadIdx.x == 0) {
        memset(&intBuf[writeIndex], 0, int_buf_size - writeIndex);
    }
}