#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitonicSort2(int *inputArray, const unsigned int stage, const unsigned int passOfStage, const unsigned int width) {
    int4 *theArray = (int4 *)inputArray;
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    int4 srcLeft, srcRight, mask;
    int4 imask10 = make_int4(0, 0, -1, -1);
    int4 imask11 = make_int4(0, -1, 0, -1);

    const unsigned int dir = 0;
    if (stage > 0) {
        if (passOfStage > 0) { // Upper level pass, exchange between two fours
            size_t r = 1 << (passOfStage - 1);
            size_t lmask = r - 1;
            size_t left = ((i >> (passOfStage - 1)) << passOfStage) + (i & lmask);
            size_t right = left + r;

            srcLeft = theArray[left];
            srcRight = theArray[right];

            mask.x = srcLeft.x < srcRight.x;
            mask.y = srcLeft.y < srcRight.y;
            mask.z = srcLeft.z < srcRight.z;
            mask.w = srcLeft.w < srcRight.w;

            int4 imin, imax;
            imin.x = (srcLeft.x & mask.x) | (srcRight.x & ~mask.x);
            imin.y = (srcLeft.y & mask.y) | (srcRight.y & ~mask.y);
            imin.z = (srcLeft.z & mask.z) | (srcRight.z & ~mask.z);
            imin.w = (srcLeft.w & mask.w) | (srcRight.w & ~mask.w);

            imax.x = (srcLeft.x & ~mask.x) | (srcRight.x & mask.x);
            imax.y = (srcLeft.y & ~mask.y) | (srcRight.y & mask.y);
            imax.z = (srcLeft.z & ~mask.z) | (srcRight.z & mask.z);
            imax.w = (srcLeft.w & ~mask.w) | (srcRight.w & mask.w);

            if (((i >> (stage - 1)) & 1) ^ dir) {
                theArray[left] = imin;
                theArray[right] = imax;
            } else {
                theArray[right] = imin;
                theArray[left] = imax;
            }
        } else { // Last pass, sort inside one four
            srcLeft = theArray[i];
            srcRight = make_int4(srcLeft.z, srcLeft.w, srcLeft.x, srcLeft.y);

            mask.x = (srcLeft.x < srcRight.x) ^ imask10.x;
            mask.y = (srcLeft.y < srcRight.y) ^ imask10.y;
            mask.z = (srcLeft.z < srcRight.z) ^ imask10.z;
            mask.w = (srcLeft.w < srcRight.w) ^ imask10.w;

            bool condition = ((i >> stage) & 1) ^ dir;
            int4 tempLeft, tempRight;
            
            tempLeft.x = (srcLeft.x & (condition ? mask.x : ~mask.x)) | (srcRight.x & (condition ? ~mask.x : mask.x));
            tempLeft.y = (srcLeft.y & (condition ? mask.y : ~mask.y)) | (srcRight.y & (condition ? ~mask.y : mask.y));
            tempLeft.z = (srcLeft.z & (condition ? mask.z : ~mask.z)) | (srcRight.z & (condition ? ~mask.z : mask.z));
            tempLeft.w = (srcLeft.w & (condition ? mask.w : ~mask.w)) | (srcRight.w & (condition ? ~mask.w : mask.w));

            srcRight = make_int4(tempLeft.y, tempLeft.x, tempLeft.w, tempLeft.z);
            mask.x = (tempLeft.x < srcRight.x) ^ imask11.x;
            mask.y = (tempLeft.y < srcRight.y) ^ imask11.y;
            mask.z = (tempLeft.z < srcRight.z) ^ imask11.z;
            mask.w = (tempLeft.w < srcRight.w) ^ imask11.w;

            theArray[i].x = (tempLeft.x & (condition ? mask.x : ~mask.x)) | (srcRight.x & (condition ? ~mask.x : mask.x));
            theArray[i].y = (tempLeft.y & (condition ? mask.y : ~mask.y)) | (srcRight.y & (condition ? ~mask.y : mask.y));
            theArray[i].z = (tempLeft.z & (condition ? mask.z : ~mask.z)) | (srcRight.z & (condition ? ~mask.z : mask.z));
            theArray[i].w = (tempLeft.w & (condition ? mask.w : ~mask.w)) | (srcRight.w & (condition ? ~mask.w : mask.w));
        }
    } else { // First stage, sort inside one four
        int4 imask0 = make_int4(0, -1, -1, 0);
        srcLeft = theArray[i];
        srcRight = make_int4(srcLeft.y, srcLeft.x, srcLeft.w, srcLeft.z);

        mask.x = (srcLeft.x < srcRight.x) ^ imask0.x;
        mask.y = (srcLeft.y < srcRight.y) ^ imask0.y;
        mask.z = (srcLeft.z < srcRight.z) ^ imask0.z;
        mask.w = (srcLeft.w < srcRight.w) ^ imask0.w;

        bool direction = (i & 1) ^ dir;
        int4 tempLeft;
        
        tempLeft.x = (srcLeft.x & (dir ? mask.x : ~mask.x)) | (srcRight.x & (dir ? ~mask.x : mask.x));
        tempLeft.y = (srcLeft.y & (dir ? mask.y : ~mask.y)) | (srcRight.y & (dir ? ~mask.y : mask.y));
        tempLeft.z = (srcLeft.z & (dir ? mask.z : ~mask.z)) | (srcRight.z & (dir ? ~mask.z : mask.z));
        tempLeft.w = (srcLeft.w & (dir ? mask.w : ~mask.w)) | (srcRight.w & (dir ? ~mask.w : mask.w));

        srcRight = make_int4(tempLeft.z, tempLeft.w, tempLeft.x, tempLeft.y);
        mask.x = (tempLeft.x < srcRight.x) ^ imask10.x;
        mask.y = (tempLeft.y < srcRight.y) ^ imask10.y;
        mask.z = (tempLeft.z < srcRight.z) ^ imask10.z;
        mask.w = (tempLeft.w < srcRight.w) ^ imask10.w;

        tempLeft.x = (tempLeft.x & (direction ? mask.x : ~mask.x)) | (srcRight.x & (direction ? ~mask.x : mask.x));
        tempLeft.y = (tempLeft.y & (direction ? mask.y : ~mask.y)) | (srcRight.y & (direction ? ~mask.y : mask.y));
        tempLeft.z = (tempLeft.z & (direction ? mask.z : ~mask.z)) | (srcRight.z & (direction ? ~mask.z : mask.z));
        tempLeft.w = (tempLeft.w & (direction ? mask.w : ~mask.w)) | (srcRight.w & (direction ? ~mask.w : mask.w));

        srcRight = make_int4(tempLeft.y, tempLeft.x, tempLeft.w, tempLeft.z);
        mask.x = (tempLeft.x < srcRight.x) ^ imask11.x;
        mask.y = (tempLeft.y < srcRight.y) ^ imask11.y;
        mask.z = (tempLeft.z < srcRight.z) ^ imask11.z;
        mask.w = (tempLeft.w < srcRight.w) ^ imask11.w;

        theArray[i].x = (tempLeft.x & (direction ? mask.x : ~mask.x)) | (srcRight.x & (direction ? ~mask.x : mask.x));
        theArray[i].y = (tempLeft.y & (direction ? mask.y : ~mask.y)) | (srcRight.y & (direction ? ~mask.y : mask.y));
        theArray[i].z = (tempLeft.z & (direction ? mask.z : ~mask.z)) | (srcRight.z & (direction ? ~mask.z : mask.z));
        theArray[i].w = (tempLeft.w & (direction ? mask.w : ~mask.w)) | (srcRight.w & (direction ? ~mask.w : mask.w));
    }
}