#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CompressionKernel(int dimensionalityd, unsigned long long *cbufd, unsigned char *dbufd, int *cutd, int *offd) {
    register int offset, code, bcount, tmp, off, beg, end, lane, warp, iindex, lastidx, start, term;
    register unsigned long long diff, prev;
    __shared__ int ibufs[32 * (3 * WARPSIZE / 2)]; // shared space for prefix sum

    // Calculate lane and warp
    lane = threadIdx.x & 31;
    warp = (threadIdx.x + blockIdx.x * blockDim.x) / WARPSIZE;

    // Setup shared memory indices
    iindex = threadIdx.x / WARPSIZE * (3 * WARPSIZE / 2) + lane;
    ibufs[iindex] = 0;
    iindex += WARPSIZE / 2;
    lastidx = (threadIdx.x / WARPSIZE + 1) * (3 * WARPSIZE / 2) - 1;

    // Determine chunk boundaries
    start = (warp > 0) ? cutd[warp - 1] : 0;
    term = cutd[warp];
    off = ((start + 1) / 2 * 17);
    offset = WARPSIZE - (dimensionalityd - lane % dimensionalityd) - lane;

    prev = 0;
    for (int i = start + lane; i < term; i += WARPSIZE) {
        // Calculate delta and leading zeros
        diff = cbufd[i] - prev;
        code = (diff >> 60) & 8;
      
        if (code != 0) {
            diff = -diff;
        }
        
        bcount = 8 - (__clzll(diff) >> 3);
        bcount = (bcount == 2) ? 3 : bcount; // Encode 6 lead-zero bytes as 5

        // Prefix sum using warp-synchronous operations
        ibufs[iindex] = bcount;
        #pragma unroll
        for(int d = 1; d < 32; d *= 2) {
            int n = __shfl_up_sync(0xFFFFFFFF, ibufs[iindex], d);
            if(lane >= d) {
                ibufs[iindex] += n;
            }
        }

        // Write non-zero bytes to compressed buffer
        beg = off + (WARPSIZE / 2) + ibufs[iindex - 1];
        end = beg + bcount;
        for (; beg < end; beg++) {
            dbufd[beg] = diff;
            diff >>= 8;
        }

        if (bcount >= 3) bcount--;
        tmp = ibufs[lastidx];
        code |= bcount;
        ibufs[iindex] = code;
      
        // Write out half-bytes
        if ((lane & 1) != 0) {
            dbufd[off + (lane >> 1)] = ibufs[iindex - 1] | (code << 4);
        }
      
        off += tmp + (WARPSIZE / 2);

        // Update prediction value
        prev = cbufd[i + offset];
    }

    // Store total bytes of compressed output
    if (lane == 31) offd[warp] = off;
}