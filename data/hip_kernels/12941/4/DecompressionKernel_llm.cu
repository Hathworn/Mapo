#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DecompressionKernel(int dimensionalityd, unsigned char *compressed_data_buffer_in, int *chunk_boundaries_buffer_in, unsigned long long *uncompressed_data_buffer_out) {
    register int offset, code, bcount, off, beg, end, lane, warp, iindex, lastidx, start, term;
    register unsigned long long diff, prev;
    __shared__ int ibufs[32 * (3 * WARPSIZE / 2)];

    lane = threadIdx.x & 31; // Thread index within warp
    iindex = threadIdx.x / WARPSIZE * (3 * WARPSIZE / 2) + lane;
    ibufs[iindex] = 0;
    iindex += WARPSIZE / 2;
    lastidx = (threadIdx.x / WARPSIZE + 1) * (3 * WARPSIZE / 2) - 1;
    warp = (threadIdx.x + blockIdx.x * blockDim.x) / WARPSIZE;
    offset = WARPSIZE - (dimensionalityd - lane % dimensionalityd) - lane;

    start = (warp > 0) ? chunk_boundaries_buffer_in[warp - 1] : 0;
    term = chunk_boundaries_buffer_in[warp];
    off = ((start + 1) / 2 * 17);

    prev = 0;
    for (int i = start + lane; i < term; i += WARPSIZE) {
        if ((lane & 1) == 0) {
            code = compressed_data_buffer_in[off + (lane >> 1)];
            ibufs[iindex] = code & 0xf; // Store lower nibble
            ibufs[iindex + 1] = code >> 4; // Store upper nibble
        }
        __syncthreads(); // Synchronize threads within block

        bcount = ibufs[iindex] & 7;
        if (bcount >= 2) bcount++;

        ibufs[iindex] = bcount;
        __syncthreads(); // Synchronize for prefix sum

        // Parallel prefix sum
        if (lane >= 1) ibufs[iindex] += ibufs[iindex - 1];
        if (lane >= 2) ibufs[iindex] += ibufs[iindex - 2];
        if (lane >= 4) ibufs[iindex] += ibufs[iindex - 4];
        if (lane >= 8) ibufs[iindex] += ibufs[iindex - 8];
        if (lane >= 16) ibufs[iindex] += ibufs[iindex - 16];
        
        __syncthreads(); // Synchronize for data read

        beg = off + ibufs[iindex - 1];
        off += ibufs[lastidx];
        end = beg + bcount - 1;
        diff = 0;
        for (; beg <= end; end--) {
            diff = (diff << 8) | compressed_data_buffer_in[end];
        }

        if ((ibufs[iindex] & 8) != 0) {
            diff = -diff;
        }

        uncompressed_data_buffer_out[i] = prev + diff;
        prev = uncompressed_data_buffer_out[i + offset];
    }
}