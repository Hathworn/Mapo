#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void move(uint8_t *buf, uint32_t dest, uint32_t source, uint16_t bytesEach, const bool wipe) {
    extern __shared__ uint8_t sharedMemT[];
    const uint32_t i = threadIdx.x;
    uint8_t *src = &buf[source + i * bytesEach];
    uint8_t *d = &buf[dest + i * bytesEach];

    for (uint16_t j = 0; j < bytesEach; j++) {
        sharedMemT[i * bytesEach + j] = src[j];  // Read data from global to shared
        if (wipe) {
            src[j] = 0;  // Zero-out the source if wipe is true
        }
    }

    __syncthreads();

    for (uint16_t j = 0; j < bytesEach; j++) {
        d[j] = sharedMemT[i * bytesEach + j];  // Write data from shared to global
    }
}