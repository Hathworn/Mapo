#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate thread's unique id and total number of threads
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    const uint numEls = tgtWidth * tgtHeight;

    // Loop over elements with stride equal to the total number of threads
    for (uint i = idx; i < numEls; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;

        // Compute source indices using modulo to tile as necessary
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Assign tiled element to the target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}