#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            // Use warp reduction to calculate numMax more efficiently
            int numMax = 0;
            for (int i = threadIdx.x; i < numOut; i += blockDim.x) {
                numMax += (probs[i * numCases + tx] == maxp) ? 1 : 0;
            }

            // Use shared memory for intermediate results within the block
            __shared__ int sharedNumMax;
            if (threadIdx.x == 0) sharedNumMax = 0;
            __syncthreads();

            // Atomic add to accumulate results from each thread
            atomicAdd(&sharedNumMax, numMax);
            __syncthreads();

            if (threadIdx.x == 0) {
                correctProbs[tx] = 1.0f / float(sharedNumMax);
            }
        }
    }
}