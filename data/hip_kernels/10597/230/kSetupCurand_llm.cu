#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate the unique thread index based on block and thread index
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    // Initialize the hiprand state for each thread with different sequence numbers
    hiprand_init(seed, tidx, 0, &state[tidx]);
}