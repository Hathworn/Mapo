#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;
    
    // Efficient loop unrolling and read in batches
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        sum += a[eidx] * b[eidx];
    }
    
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Optimized warp reduction
    for (uint s = DP_BLOCKSIZE / 2; s > 32; s >>= 1) {
        if (threadIdx.x < s) {
            shmem[threadIdx.x] += shmem[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Warp-level reduction with volatile
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        if (threadIdx.x < 16) mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        if (threadIdx.x < 8) mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        if (threadIdx.x < 4) mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        if (threadIdx.x < 2) mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        if (threadIdx.x < 1) mysh[threadIdx.x] += mysh[threadIdx.x + 1];
        if (threadIdx.x == 0) {
            target[blockIdx.x] = mysh[0];
        }
    }
}