#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use blockIdx.x as index and ensure stride over gridDim.x for larger datasets
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        accuracy[idx] /= N;
    }
}