#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuSearchDoublet(const int* nSpM, const float* spMmat, const int* nSpB, const float* spBmat, const int* nSpT, const float* spTmat, const float* deltaRMin, const float* deltaRMax, const float* cotThetaMax, const float* collisionRegionMin, const float* collisionRegionMax, int* nSpMcomp, int* nSpBcompPerSpM_Max, int* nSpTcompPerSpM_Max, int* nSpBcompPerSpM, int* nSpTcompPerSpM, int* McompIndex, int* BcompIndex, int* tmpBcompIndex, int* TcompIndex, int* tmpTcompIndex) {
    extern __shared__ float sharedMem[];
    int* mPos = (int*)sharedMem;
    int* isMcompat = (int*)&mPos[1];

    if (threadIdx.x == 0) {
        *isMcompat = false;
    }
    __syncthreads();

    float rM = spMmat[blockIdx.x + (*nSpM) * 3];
    float zM = spMmat[blockIdx.x + (*nSpM) * 2];

    int offset = 0;
    const int nSpB_val = *nSpB;
    const int nSpT_val = *nSpT;
    const float deltaRMin_val = *deltaRMin;
    const float deltaRMax_val = *deltaRMax;
    const float cotThetaMax_val = *cotThetaMax;
    const float collisionRegionMin_val = *collisionRegionMin;
    const float collisionRegionMax_val = *collisionRegionMax;

    while (offset < max(nSpB_val, nSpT_val)) {
        if (threadIdx.x + offset < nSpB_val) {
            float rB = spBmat[threadIdx.x + offset + nSpB_val * 3];
            float zB = spBmat[threadIdx.x + offset + nSpB_val * 2];
            float deltaR = rM - rB;
            bool isBcompat = (deltaR <= deltaRMax_val && deltaR >= deltaRMin_val);
            float cotTheta = (zM - zB) / deltaR;
            isBcompat &= (fabsf(cotTheta) <= cotThetaMax_val);
            float zOrigin = zM - rM * cotTheta;
            isBcompat &= (zOrigin >= collisionRegionMin_val && zOrigin <= collisionRegionMax_val);

            if (isBcompat) {
                int bPos = atomicAdd(&nSpBcompPerSpM[blockIdx.x], 1);
                tmpBcompIndex[bPos + nSpB_val * blockIdx.x] = threadIdx.x + offset;
            }
        }

        if (threadIdx.x + offset < nSpT_val) {
            float rT = spTmat[threadIdx.x + offset + nSpT_val * 3];
            float zT = spTmat[threadIdx.x + offset + nSpT_val * 2];
            float deltaR = rT - rM;
            bool isTcompat = (deltaR >= deltaRMin_val && deltaR <= deltaRMax_val);
            float cotTheta = (zT - zM) / deltaR;
            isTcompat &= (fabsf(cotTheta) <= cotThetaMax_val);
            float zOrigin = zM - rM * cotTheta;
            isTcompat &= (zOrigin >= collisionRegionMin_val && zOrigin <= collisionRegionMax_val);

            if (isTcompat) {
                int tPos = atomicAdd(&nSpTcompPerSpM[blockIdx.x], 1);
                tmpTcompIndex[tPos + nSpT_val * blockIdx.x] = threadIdx.x + offset;
            }
        }

        offset += blockDim.x;
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        if (nSpBcompPerSpM[blockIdx.x] > 0 && nSpTcompPerSpM[blockIdx.x] > 0) {
            *mPos = atomicAdd(nSpMcomp, 1);
            *isMcompat = true;
            McompIndex[*mPos] = blockIdx.x;

            atomicMax(nSpBcompPerSpM_Max, nSpBcompPerSpM[blockIdx.x]);
            atomicMax(nSpTcompPerSpM_Max, nSpTcompPerSpM[blockIdx.x]);
        }
    }

    __syncthreads();

    if (*isMcompat == true) {
        offset = 0;
        const int nSpBcomp = nSpBcompPerSpM[blockIdx.x];
        const int nSpTcomp = nSpTcompPerSpM[blockIdx.x];
        while (offset < max(nSpBcomp, nSpTcomp)) {
            if (threadIdx.x + offset < nSpBcomp) {
                BcompIndex[threadIdx.x + offset + nSpB_val * (*mPos)] = tmpBcompIndex[threadIdx.x + offset + nSpB_val * blockIdx.x];
            }
            if (threadIdx.x + offset < nSpTcomp) {
                TcompIndex[threadIdx.x + offset + nSpT_val * (*mPos)] = tmpTcompIndex[threadIdx.x + offset + nSpT_val * blockIdx.x];
            }
            offset += blockDim.x;
        }
    }
}