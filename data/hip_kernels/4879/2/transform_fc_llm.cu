#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define CUDA_CHECK_ERROR

#define CudaSafeCall(err) __CudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError() __CudaCheckError(__FILE__, __LINE__)

__global__ void transform_fc(float *input, const float *raw_input, const int width, const int channels)
{
    int thread_id = threadIdx.x;
    int size = width * width;
    int total_size = size * channels;
    
    // Loop unrolling for better performance
    for (int s = thread_id; s < total_size; s += channels)
    {
        input[s] = raw_input[(s % size) * channels + thread_id];
    }

    // Use first thread of the first block to set the last element
    if (blockIdx.x == 0 && thread_id == 0)
    {
        input[total_size] = 1;
    }
}