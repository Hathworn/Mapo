#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define CUDA_CHECK_ERROR

#define CudaSafeCall(err) __CudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError() __CudaCheckError(__FILE__, __LINE__)

__global__ void transform_image(float *input, const float *raw_input, const int width, const int channels)
{
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int start_i = thread_id / width - 1;
    int start_j = thread_id % width - 1;
    int per_channel_width = width * width;
    int hidden_width = 3 * 3 * channels + 1;
    int global_offset = thread_id * hidden_width;

    // Unroll the loop for better performance and reduce branching
    #pragma unroll
    for (int c = 0; c < channels; c++) {
        int offset = 0;
        for (int i = start_i; i < start_i + 3; i++) {
            bool valid_i = (i >= 0 && i < width); // Cache comparison result
            for (int j = start_j; j < start_j + 3; j++) {
                if (valid_i && j >= 0 && j < width) { // Use cached result
                    input[global_offset + c * 9 + offset] = raw_input[c * per_channel_width + i * width + j];
                }
                offset++;
            }
        }
    }
    input[(thread_id + 1) * hidden_width - 1] = 1;
}