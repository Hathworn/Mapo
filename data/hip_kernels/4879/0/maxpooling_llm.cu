#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define CUDA_CHECK_ERROR
#define CudaSafeCall(err) __CudaSafeCall(err, __FILE__, __LINE__)
#define CudaCheckError() __CudaCheckError(__FILE__, __LINE__)

__global__ void maxpooling(float *output, const float *input, const int width, const int channels)
{
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int new_width = width / 2;
    
    // Calculate 2D coordinates directly to reduce redundant calculations
    int row = thread_id / new_width;
    int col = thread_id % new_width;
    
    // Calculate top-left corner index of the 2x2 window
    int i = row * 2;
    int j = col * 2;
    
    int index = i * width + j;
    
    for (int c = 0; c < channels; c++) {
        // Initialize max with the first element
        float max = input[index * channels + c];
        
        // Use fmaxf to simplify comparisons
        max = fmaxf(max, input[(index + 1) * channels + c]);
        max = fmaxf(max, input[(index + width) * channels + c]);
        max = fmaxf(max, input[(index + width + 1) * channels + c]);
        
        output[thread_id * channels + c] = max;
    }
}