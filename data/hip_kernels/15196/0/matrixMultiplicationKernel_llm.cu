#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMultiplicationKernel(float* A, float* B, float* C, int N) {
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize temporary sum for each thread
    float tmpSum = 0;

    // Check if within matrix bounds
    if (ROW < N && COL < N) {
        // Use shared memory for sub-matrices
        __shared__ float sharedA[16][16];
        __shared__ float sharedB[16][16];

        for (int m = 0; m < (N + 15) / 16; ++m) {
            // Load data into shared memory
            if (m * 16 + threadIdx.x < N && ROW < N)
                sharedA[threadIdx.y][threadIdx.x] = A[ROW * N + m * 16 + threadIdx.x];
            else
                sharedA[threadIdx.y][threadIdx.x] = 0.0;

            if (m * 16 + threadIdx.y < N && COL < N)
                sharedB[threadIdx.y][threadIdx.x] = B[(m * 16 + threadIdx.y) * N + COL];
            else
                sharedB[threadIdx.y][threadIdx.x] = 0.0;

            // Synchronize to ensure all threads have loaded their sub-matrices
            __syncthreads();

            // Perform multiplication on the sub-matrix
            for (int e = 0; e < 16; ++e)
                tmpSum += sharedA[threadIdx.y][e] * sharedB[e][threadIdx.x];

            // Synchronize again to ensure all threads have finished computing
            __syncthreads();
        }
    }

    // Write back the result to the global matrix
    if (ROW < N && COL < N)
        C[ROW * N + COL] = tmpSum;
}