#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void final_map_index_to_prefix(uint8_t *hashes, uint32_t *sort_indices, uint32_t *off_map, uint32_t *comb_count, uint32_t *comb_sum, uint32_t *comb_prefix, uint32_t r, uint32_t size) 
{
    uint32_t t_index = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure the thread index is within bounds
    if (t_index >= size) return;

    uint32_t index = sort_indices[t_index];
    unsigned char *hash = hashes + index * 30 + r * 3;  // Eliminate redundant sizeof

    // Compactly combine bytes into a 64-bit key
    uint64_t key = ((uint64_t)hash[0] << 40) | ((uint64_t)hash[1] << 32) | (hash[2] << 24) |
                   (hash[3] << 16) | (hash[4] << 8)  | hash[5];

    uint32_t lookup_idx = off_map[t_index];
    uint64_t count = comb_count[lookup_idx];
    uint64_t sum = comb_sum[lookup_idx];

    // Use a single assignment to avoid potential race conditions
    uint64_t start = sum - count;
    for (uint64_t i = start; i < sum; i++) {
        comb_prefix[i] = lookup_idx;
    }
}