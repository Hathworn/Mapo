#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Copyright Douglas Goddard 2016
// Licensed under the MIT license

__global__ void remap_reduction(uint32_t *d_reduction, uint32_t *d_mapping, uint32_t *old_d_ij_buf, uint32_t sum_prev_size, uint32_t prev_size, uint32_t *new_d_ij_buf, uint32_t new_size)
{
    uint32_t t_index = blockDim.x * blockIdx.x + threadIdx.x;
    if (t_index >= prev_size) return;  // Early exit for out-of-bounds threads
    if (!d_reduction[t_index]) return; // Skip unnecessary computations
    
    uint32_t index = d_mapping[t_index];
    uint32_t offset = 2 * sum_prev_size; // Precompute common offset
    uint32_t i = old_d_ij_buf[offset + t_index];
    uint32_t j = old_d_ij_buf[offset + prev_size + t_index];

    // sort pairs in first round
    if (!sum_prev_size && j < i) {
        uint32_t temp = i; // Use a temporary variable for clarity and readability
        i = j;
        j = temp;
    }

    new_d_ij_buf[offset + index] = i;
    new_d_ij_buf[offset + new_size + index] = j;
}