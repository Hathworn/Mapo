#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void get_sort_keys(unsigned char* d_hashes, uint32_t *d_sort_keys, uint32_t *d_sort_indices, uint32_t r, uint32_t num_keys) 
{
    uint32_t index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < num_keys) {
        d_sort_indices[index] = index;

        // Optimize memory access by reducing pointer arithmetic and using local variable
        unsigned char* input = d_hashes + index * 30 + 3 * r;
        uint32_t byte0 = input[0];
        uint32_t byte1 = input[1];
        uint32_t byte2 = input[2];

        // Create sort_key with direct bit shifting and OR operation
        uint32_t sort_key = (byte0 << 16) | (byte1 << 8) | byte2;
        d_sort_keys[index] = sort_key;
    }
}