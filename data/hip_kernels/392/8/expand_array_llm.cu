#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expand_array(unsigned char *d_in, unsigned char *d_out) {
    uint32_t offset = blockDim.x * blockIdx.x + threadIdx.x;

    // Pre-calculate index based on the offset
    unsigned char *input = &d_in[offset * 5];
    unsigned char *output = &d_out[offset * 6];

    // Efficient memory access with minimal operations
    unsigned char in0 = input[0];
    unsigned char in1 = input[1];
    unsigned char in2 = input[2];

    output[0] = in0 >> 4;
    output[1] = (in0 << 4) | (in1 >> 4);
    output[2] = (in1 << 4) | (in2 >> 4);
    output[3] = in2 & 0xF;
    output[4] = input[3];
    output[5] = input[4];
}