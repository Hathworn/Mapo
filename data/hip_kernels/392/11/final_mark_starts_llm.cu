#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void final_mark_starts(uint8_t *hashes, uint32_t *sort_indices, uint32_t *off_map, uint32_t r, uint32_t hash_count) {
    uint32_t t_index = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure thread index is within bounds
    if(t_index >= hash_count) return;

    uint32_t t_prev_index = (t_index == 0) ? (hash_count - 1) : (t_index - 1);

    uint32_t index = sort_indices[t_index];
    uint32_t prev_index = sort_indices[t_prev_index];

    // Simplified pointer arithmetic
    unsigned char* hash = hashes + index * 30 + r * 3;
    unsigned char* prev_hash = hashes + prev_index * 30 + r * 3;

    // Use bitwise operations to create key
    uint64_t key = *reinterpret_cast<const uint64_t*>(hash) & 0xFFFFFF000000;
    uint64_t prev_key = *reinterpret_cast<const uint64_t*>(prev_hash) & 0xFFFFFF000000;

    // Identify off_map if keys differ
    if(key != prev_key) {
        off_map[t_index] = 1;
    }
}