#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global memory access and reduce divergence in conditional checks
__global__ void map_index_to_prefix(uint8_t *hashes, uint32_t *sort_indices, uint32_t *comb_count, uint32_t *comb_sum, uint32_t *comb_prefix, uint32_t r, uint32_t size)
{
    uint32_t t_index = blockDim.x * blockIdx.x + threadIdx.x;

    if (t_index < size) {
        // Coalesce global memory access
        uint32_t index = sort_indices[t_index];
        unsigned char *hash = hashes + (index * 30 + r * 3) * sizeof(unsigned char);
        uint32_t key = (hash[0] << 16) | (hash[1] << 8) | hash[2];

        uint32_t count = comb_count[key];
        uint32_t sum = comb_sum[key];
        
        // Unroll the loop for better performance
        #pragma unroll
        for (int i = sum - count; i < sum; i++) {
            comb_prefix[i] = key;
        }
    }
}