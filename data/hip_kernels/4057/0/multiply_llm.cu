#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCK_SIZE 16

__global__ void multiply(float *left, float *right, float *res, int dim) {

    float temp = 0;
    __shared__ float Left_shared_t[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Right_shared_t[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    for (int tileNUM = 0; tileNUM < gridDim.x; tileNUM++) {

        int i = tileNUM * BLOCK_SIZE + threadIdx.y;
        int j = tileNUM * BLOCK_SIZE + threadIdx.x;

        // Load left and right matrix into shared memory
        if (row < dim && j < dim) {
            Left_shared_t[threadIdx.y][threadIdx.x] = left[row * dim + j];
        } else {
            Left_shared_t[threadIdx.y][threadIdx.x] = 0.0f; // Boundary condition handling
        }

        if (i < dim && col < dim) {
            Right_shared_t[threadIdx.y][threadIdx.x] = right[i * dim + col];
        } else {
            Right_shared_t[threadIdx.y][threadIdx.x] = 0.0f; // Boundary condition handling
        }

        // Synchronize before computation
        __syncthreads();

        // Perform matrix multiplication on the block
        for (int k = 0; k < BLOCK_SIZE; k++) {
            temp += Left_shared_t[threadIdx.y][k] * Right_shared_t[k][threadIdx.x];
        }

        // Synchronize to ensure all threads have completed computation before reusing shared memory
        __syncthreads();
    }

    // Store result, ensure that threads do not write out of bounds
    if (row < dim && col < dim) {
        res[row * dim + col] = temp;
    }
}