#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize loop by removing unnecessary computation
    if (idx < numEls) {
        dest[idx] = a[idx] + b[idx];
    }
}