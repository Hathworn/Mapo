#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int totalThreads = gridDim.x * blockDim.x; // Calculate total number of threads

    // Utilize stride pattern to allow threads to process more elements
    for (int i = idx; i < size; i += totalThreads) {
        r[i] = x[i] * y[i] + z[i];
    }
}