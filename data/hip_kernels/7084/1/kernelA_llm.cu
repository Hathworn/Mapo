#include "hip/hip_runtime.h"
```cuda
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelA(float* r, float* x, float* y, float* z, int size)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Stride loop to ensure all elements are processed
    while (i < size) {
        r[i] = x[i] * y[i] + z[i];
        i += blockDim.x * gridDim.x;
    }
}