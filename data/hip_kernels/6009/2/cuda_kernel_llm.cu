#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_kernel(double *A, double *B, double *C, int arraySize) {
    // Calculate a unique thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within array bounds
    if (tid < arraySize) {
        // Perform addition directly
        C[tid] = A[tid] + B[tid];
    }
}