#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using threadIdx for better parallelism.
__global__ void square(int *array, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (tid < n) {
        array[tid] *= array[tid]; // Square the element
    }
}