#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use block and thread indexing for potential parallel execution and to handle larger N
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {  // Ensure only one thread performs the operation
        *accuracy /= N;
    }
}