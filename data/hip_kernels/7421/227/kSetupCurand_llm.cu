#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Compute global thread index
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Initialize hiprand for each thread with unique sequence
    hiprand_init(seed, tidx, 0, &state[tidx]);
}