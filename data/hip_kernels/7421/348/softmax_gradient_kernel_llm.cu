#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // Unroll loop for better performance
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Optimize reduction with a loop unrolling method
    if (idx == 0) {
        tmp = 0.0f;
        for (int i = 0; i < blockDim.x; i += 4) {
            if (i < blockDim.x) tmp += reduction_buffer[i];
            if (i + 1 < blockDim.x) tmp += reduction_buffer[i + 1];
            if (i + 2 < blockDim.x) tmp += reduction_buffer[i + 2];
            if (i + 3 < blockDim.x) tmp += reduction_buffer[i + 3];
        }
        reduction_buffer[0] = tmp;
    }
    __syncthreads();

    // Parallelize gradient computation
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}