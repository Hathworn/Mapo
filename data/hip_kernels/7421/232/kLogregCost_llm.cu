#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;
    
    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];
        
        labelLogProbs[tx] = __logf(labelp);

        // Early exit if label probability does not match max probability
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            // Calculate number of labels with maximum probability
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp);
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}