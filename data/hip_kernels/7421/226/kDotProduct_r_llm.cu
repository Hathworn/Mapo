#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float localSum = 0; // Use a register to hold local accumulation
    if (eidx < numElements) { // Correct boundary condition check
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            localSum += a[eidx] * b[eidx];
        }
    }
    shmem[threadIdx.x] = localSum;
    __syncthreads();
    
    // Unroll the loop for reduction
    if (threadIdx.x < 256) shmem[threadIdx.x] += shmem[threadIdx.x + 256];
    __syncthreads();
    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    __syncthreads();
    
    // Reduce within a warp without using __syncthreads(), utilize warp intrinsic
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0]; // Use the first element of shared memory to store result
    }
}