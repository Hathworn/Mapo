#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use blockIdx.x to allow for multiple blocks processing different parts of accuracy
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index == 0) {
        // Ensure that only one thread performs the division
        *accuracy /= N;
    }
}