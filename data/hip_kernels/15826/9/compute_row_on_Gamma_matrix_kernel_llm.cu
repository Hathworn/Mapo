#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel optimized using threads within blocks for parallel computation
__global__ void compute_row_on_Gamma_matrix_kernel(int row_index, int vertex_index, int* indices, double* exp_V, double* N_ptr, int LD_N, double* G_ptr, int LD_G, double* row_ptr, int incr) {
    int l = blockIdx.x * blockDim.x + threadIdx.x; // Unique thread index across grid
    if (l >= vertex_index) return; // Early exit condition for out-of-bounds

    int i_index = indices[row_index];
    int j_index = indices[l];
    double delta;

    if (j_index < vertex_index) {
        delta = i_index == j_index ? 1.0 : 0.0;
        row_ptr[l * incr] = (N_ptr[i_index + LD_N * j_index] * exp_V[l] - delta) / (exp_V[l] - 1.0);
    } else {
        row_ptr[l * incr] = G_ptr[i_index + LD_G * (j_index - vertex_index)];
    }
}