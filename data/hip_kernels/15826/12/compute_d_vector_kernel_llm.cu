#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_d_vector_kernel(int N_i, int* d_ind, double* d_ptr, int* p_ptr, double* N_ptr, int N_ld) {
    int I = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure I is within the bounds
    if (I >= N_i) return;

    int idx = d_ind[I];
    int index = p_ptr[idx];

    // Inline multiplication to reduce indexing overhead
    d_ptr[idx] = 1.0 / N_ptr[index * (N_ld + 1)];
}