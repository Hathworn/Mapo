#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void read_G_matrix_kernel(int S, int vertex_index, int* i_index, int* j_index, bool* is_Bennett, double* exp_Vj, double* N_ptr, int LD_N, double* G_ptr, int LD_G, double* result_ptr, int incr) {
    int l = threadIdx.x + blockIdx.x * blockDim.x; // Parallelize across threads and blocks

    if (l < S) { // Boundary check for l to prevent out-of-bounds access
        double result, delta;
        
        int j_idx = j_index[l];
        int i_idx = i_index[l];
        double exp_val = exp_Vj[l];
        
        if (j_idx < vertex_index) {
            delta = (i_idx == j_idx) ? 1.0 : 0.0;
            result = (N_ptr[i_idx + LD_N * j_idx] * exp_val - delta) / (exp_val - 1.0);
        } else {
            result = G_ptr[i_idx + LD_G * (j_idx - vertex_index)];
        }

        result_ptr[l * incr] = is_Bennett[l] ? 0.0 : result;
    }
}