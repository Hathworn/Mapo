#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function for vector addition
__global__ void sum(int *A , int *B, int *C, long long N)
{
    long long idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unrolling the loop to improve memory access efficiency
    #pragma unroll 4
    for (long long offset = idx; offset < N; offset += blockDim.x * gridDim.x)
    {
        C[offset] = A[offset] + B[offset];
    }
}