#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_destinations(hiprandState *state, int n, const uint32_t *sources, uint32_t *destinations) {
    int first = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory to store sources
    extern __shared__ uint32_t shared_sources[];
    if (first < n) {
        shared_sources[threadIdx.x] = sources[first];
    }
    __syncthreads();

    hiprandState local_state = state[first];
    for (int id = first; id < n; id += stride) {
        // Access shared memory
        destinations[id] = shared_sources[hiprand(&local_state) % blockDim.x];
    }

    state[first] = local_state;
}