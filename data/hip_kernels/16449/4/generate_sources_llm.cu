#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generate_sources(hiprandState *state, int n, uint32_t *verts) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    // Loop through in a more memory-friendly way
    for (; id < n; id += stride) {
        // Load the state only once per loop iteration
        hiprandState local_state = state[id];
        verts[id] = hiprand(&local_state);
        state[id] = local_state;
    }
}