#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void myset(unsigned long long *p, unsigned long long v, long long n) {
    // Cache block and thread id computation
    const long long tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a loop to allow threads to handle multiple elements
    for (long long i = tid; i < n; i += blockDim.x * gridDim.x) {
        p[i] = v;
    }
}