#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeCoarseGrained(float *odata, float *idata, int width, int height)
{
    __shared__ float block[TILE_DIM][TILE_DIM+1];

    // Calculate input and output indices
    int xIndex_in = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex_in = blockIdx.y * TILE_DIM + threadIdx.y;
    int index_in = xIndex_in + (yIndex_in)*width;

    int xIndex_out = blockIdx.y * TILE_DIM + threadIdx.x;
    int yIndex_out = blockIdx.x * TILE_DIM + threadIdx.y;
    int index_out = xIndex_out + (yIndex_out)*height;

    // Coalesced global memory read
    for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
        block[threadIdx.y+i][threadIdx.x] = idata[index_in+i*width];
    }

    __syncthreads();

    // Coalesced global memory write
    for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
        odata[index_out+i*height] = block[threadIdx.y+i][threadIdx.x];
    }
}
```
