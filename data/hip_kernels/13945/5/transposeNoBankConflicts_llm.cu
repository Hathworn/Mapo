#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNoBankConflicts(float *odata, float *idata, int width, int height)
{
    __shared__ float tile[TILE_DIM][TILE_DIM+1];

    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
    int index_in = xIndex + (yIndex)*width;

    xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
    yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
    int index_out = xIndex + (yIndex)*height;

    // Load data into shared memory efficiently
    if (xIndex < width && yIndex < height) {
        for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
            if (yIndex + i < height) {
                tile[threadIdx.y + i][threadIdx.x] = idata[index_in + i * width];
            }
        }
    }

    __syncthreads();

    // Write data from shared memory to output array efficiently
    if (xIndex < height && yIndex < width) {
        for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
            if (yIndex + i < width) {
                odata[index_out + i * height] = tile[threadIdx.x][threadIdx.y + i];
            }
        }
    }
}