#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copySharedMem(float *odata, float *idata, int width, int height)
{
    __shared__ float tile[TILE_DIM][TILE_DIM];

    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
    
    int index = xIndex + width * yIndex;

    // Preload data into shared memory with bounds checking
    if (xIndex < width && yIndex < height)
    {
        for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
        {
            if (yIndex + i < height) // Ensure index is within bounds
            {
                tile[threadIdx.y + i][threadIdx.x] = idata[index + i * width];
            }
        }
    }
    
    __syncthreads();
    
    // Write shared memory data to global memory with bounds checking
    if (xIndex < height && yIndex < width)
    {
        for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
        {
            if (yIndex + i < width) // Ensure index is within bounds
            {
                odata[index + i * width] = tile[threadIdx.y + i][threadIdx.x];
            }
        }
    }
}