#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNaive(float *odata, float *idata, int width, int height)
{
    // Calculate transposed block indices first
    int xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.x * TILE_DIM + threadIdx.y;

    // Calculate input and output indices
    int index_in  = yIndex + width * xIndex;
    int index_out = xIndex + height * yIndex;

    // Loop unrolling and data coalescing
    for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS)
    {
        odata[index_out + i * height] = idata[index_in + i * width];
    }
}