#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeFineGrained(float *odata, float *idata, int width, int height)
{
    __shared__ float block[TILE_DIM][TILE_DIM+1]; // Padding to avoid bank conflicts.

    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

    // Optimize: Check bounds to prevent out-of-bounds memory access.
    if (xIndex < width && yIndex < height) {
        int index = xIndex + yIndex * width;

        // Load matrix tile into shared memory
        for (int i = 0; i < TILE_DIM && (threadIdx.y + i < height); i += BLOCK_ROWS)
        {
            block[threadIdx.y + i][threadIdx.x] = idata[index + i * width];
        }

        __syncthreads();

        // Write transposed tile to output matrix
        for (int i = 0; i < TILE_DIM && (threadIdx.x + i < width); i += BLOCK_ROWS)
        {
            // Adjust column index using height here for correct transposition
            odata[(blockIdx.y * TILE_DIM + threadIdx.x) + (blockIdx.x * TILE_DIM + threadIdx.y + i) * height] = block[threadIdx.x + i][threadIdx.y];
        }
    }
}