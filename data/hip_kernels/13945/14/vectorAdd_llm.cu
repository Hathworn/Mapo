#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    // Using shared memory for faster access
    extern __shared__ float sharedA[];
    extern __shared__ float sharedB[];

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Loading data into shared memory
    if (i < numElements)
    {
        sharedA[threadIdx.x] = A[i];
        sharedB[threadIdx.x] = B[i];
    }

    __syncthreads();

    // Calculating result
    if (i < numElements)
    {
        C[i] = sharedA[threadIdx.x] + sharedB[threadIdx.x];
    }
}