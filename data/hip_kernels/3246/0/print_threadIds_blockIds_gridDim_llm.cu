#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_threadIds_blockIds_gridDim()
{
    // Use a single printf to minimize I/O traffic and improve performance
    printf("threadIdx: (%d, %d, %d), blockIdx: (%d, %d, %d), gridDim: (%d, %d, %d) \n",
           threadIdx.x, threadIdx.y, threadIdx.z,
           blockIdx.x, blockIdx.y, blockIdx.z,
           gridDim.x, gridDim.y, gridDim.z);
}