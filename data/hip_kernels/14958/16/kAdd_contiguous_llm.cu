#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Use shared memory to reduce global memory access and improve performance
    __shared__ float shared_a[256]; // Assuming blockDim.x = 256 for simplicity
    __shared__ float shared_b[256];

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        shared_a[threadIdx.x] = a[i];
        shared_b[threadIdx.x] = b[i];
        __syncthreads(); // Synchronize threads within the block

        dest[i] = shared_a[threadIdx.x] + shared_b[threadIdx.x];
        __syncthreads(); // Ensure all threads have completed writing to dest
    }
}