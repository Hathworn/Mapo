#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMulGPU(int *a, int *b, int *c) 
{
    // Use shared memory for more efficient access
    __shared__ int sharedA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int sharedB[BLOCK_SIZE][BLOCK_SIZE];
    
    int val = 0;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    for (int m = 0; m < (N + BLOCK_SIZE - 1) / BLOCK_SIZE; ++m) 
    {
        // Load data into shared memory
        if (row < N && m * BLOCK_SIZE + threadIdx.y < N)
            sharedA[threadIdx.x][threadIdx.y] = a[row * N + m * BLOCK_SIZE + threadIdx.y];
        else
            sharedA[threadIdx.x][threadIdx.y] = 0;

        if (col < N && m * BLOCK_SIZE + threadIdx.x < N)
            sharedB[threadIdx.x][threadIdx.y] = b[(m * BLOCK_SIZE + threadIdx.x) * N + col];
        else
            sharedB[threadIdx.x][threadIdx.y] = 0;
        
        __syncthreads();

        // Compute the partial results
        for (int k = 0; k < BLOCK_SIZE; ++k)
            val += sharedA[threadIdx.x][k] * sharedB[k][threadIdx.y];
        
        __syncthreads();
    }

    // Write the result to the matrix
    if (row < N && col < N) 
        c[row * N + col] = val;
}