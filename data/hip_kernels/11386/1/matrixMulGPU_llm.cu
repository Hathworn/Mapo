#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMulGPU(int* a, int* b, int* c)
{
    // Use shared memory to improve data locality.
    __shared__ int sharedA[TILE_WIDTH][TILE_WIDTH];
    __shared__ int sharedB[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Identify the row and column of the Pd element to work on.
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;
    int val = 0;

    // Loop over the a and b tiles required to compute the c element.
    for (int m = 0; m < (N - 1) / TILE_WIDTH + 1; ++m)
    {
        // Load tiles into shared memory, checking bounds.
        if (m * TILE_WIDTH + tx < N && row < N)
            sharedA[ty][tx] = a[row * N + m * TILE_WIDTH + tx];
        else
            sharedA[ty][tx] = 0;

        if (m * TILE_WIDTH + ty < N && col < N)
            sharedB[ty][tx] = b[(m * TILE_WIDTH + ty) * N + col];
        else
            sharedB[ty][tx] = 0;

        // Synchronize to make sure the sub-matrices are loaded.
        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k)
            val += sharedA[ty][k] * sharedB[k][tx];

        // Synchronize to make sure that the preceding computation is done before loading new sub-matrices in the next iteration.
        __syncthreads();
    }

    // Write the block sub-matrix to global memory, checking bounds.
    if (row < N && col < N)
        c[row * N + col] = val;
}