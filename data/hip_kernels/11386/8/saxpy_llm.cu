#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy(float scalar, float * x, float * y, int n)
{
    // Calculate the global index of the element to be processed
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) // Ensure index is within bounds
    {
        y[tid] = scalar * x[tid] + y[tid]; // Perform SAXPY operation
    }
}
```
