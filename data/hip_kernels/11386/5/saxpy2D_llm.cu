#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy2D(float scalar, float * x, float * y, int NX, int NY)
{
    // Calculate the unique thread index in the flattened 2D grid
    int index = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x;

    // Optimize by removing redundant calculations, perform SAXPY if index is within bounds
    if (index < NX * NY)
    {
        y[index] = scalar * x[index] + y[index];
    }
}