#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy2D(float scalar, float * x, float * y, int NX, int NY)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // Use a single index calculation for performance
    int index = row * NY + col;

    // Ensure computation is within bounds
    if (row < NX && col < NY) {
        y[index] = scalar * x[index] + y[index];
    }
}