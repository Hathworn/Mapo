#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matByConst(unsigned char *img, unsigned char *result, int alpha, int cols, int rows) {
    // Use shared memory to cache image data for efficiency
    __shared__ unsigned char tile[32][32]; 

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        // Load data into shared memory
        tile[threadIdx.y][threadIdx.x] = img[row * cols + col];

        // Synchronize all threads in block to ensure all data is loaded
        __syncthreads();
        
        // Perform the computation using shared memory
        result[row * cols + col] = tile[threadIdx.y][threadIdx.x] * alpha;
    }
}