#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy(int * a, int * b, int * c)
{
    // Fetch per-thread offset to process each element
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop to maximize utilization of warp
    #pragma unroll
    if (tid < N) {
        c[tid] = 2 * a[tid] + b[tid];
    }
}