#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy(float scalar, float * x, float * y)
{
    // Calculate the global thread index for processing data
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure all memory accesses are coalesced and within bounds
    if (tid < N) {
        y[tid] = scalar * x[tid] + y[tid];
    }
}