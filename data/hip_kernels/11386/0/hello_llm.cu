#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void hello()
{
    // Use threadIdx.x only to reduce redundant computations
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    // Load hello string once and reuse
    const char* helloString = "Hello from Thread";

    printf("%s %d in block %d\n", helloString, tid, bid); 
}