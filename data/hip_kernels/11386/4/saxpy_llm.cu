#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy(int * a, int * b, int * c)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Process data within the bounds
    if (tid < N) 
    {
        // Direct evaluation of SAXPY operation
        int temp_a = a[tid]; // Load a[tid] once into a register
        c[tid] = 2 * temp_a + b[tid]; // Compute and store result
    }
}