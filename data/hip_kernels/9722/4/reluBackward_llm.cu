#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize: Use 1D blocks as we only need one index calculation

extern "C"
__global__ void reluBackward(double* X, double* dout, double* ret, int rlen, int clen) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = gridDim.x * blockDim.x;

    for (int i = index; i < rlen * clen; i += totalThreads) {
        ret[i] = X[i] > 0 ? dout[i] : 0;
    }
}