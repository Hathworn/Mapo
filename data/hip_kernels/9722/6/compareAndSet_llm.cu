#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Licensed to the Apache Software Foundation (ASF) under one
* or more contributor license agreements.  See the NOTICE file
* distributed with this work for additional information
* regarding copyright ownership.  The ASF licenses this file
* to you under the Apache License, Version 2.0 (the
* "License"); you may not use this file except in compliance
* with the License.  You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing,
* software distributed under the License is distributed on an
* "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
* KIND, either express or implied.  See the License for the
* specific language governing permissions and limitations
* under the License.
*/

/**********************************
When updating a kernel or adding a new one,
please compile the ptx file and commit it:
nvcc -ptx -arch=sm_30 SystemML.cu
***********************************/


extern "C"
__global__ void compareAndSet(double* A, double* ret, int rlen, int clen, double compareVal, double tol, double ifEqualsVal, double ifLessThanVal, double ifGreaterThanVal) {
    int index = blockIdx.x * blockDim.x * clen + threadIdx.x * clen + blockIdx.y * blockDim.y + threadIdx.y;
    // Ensure index is within bounds of matrix
    if (index < rlen * clen) {
        double A_val = A[index];
        // Perform comparison and assignment
        if (fabs(A_val - compareVal) < tol) {
            ret[index] = ifEqualsVal;
        } else if (A_val < compareVal) {
            ret[index] = ifLessThanVal;
        } else {
            ret[index] = ifGreaterThanVal;
        }
    }
}