#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int n, float *a, float *b, float *sum)
{
    // Use a variable instead of repeatedly fetching it
    int totalThreads = blockDim.x * gridDim.x;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += totalThreads)
    {
        sum[i] = a[i] + b[i]; // Perform addition in coalesced memory access pattern
    }
}