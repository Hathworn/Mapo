#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMul(int *a, int *b, int *c, int size){
    // Calculate unique global thread indices
    int my_x = blockIdx.x * blockDim.x + threadIdx.x;
    int my_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check to ensure within matrix bounds
    if (my_x < size && my_y < size) {
        int local_c = 0;
        // Loop unrolling optimization could be here if we know size is divisible
        for (int i = 0; i < size; i++) {
            local_c += a[my_x * size + i] * b[i * size + my_y];
        }
        // Assign the result to the matrix c
        c[my_x * size + my_y] = local_c;
    }
}