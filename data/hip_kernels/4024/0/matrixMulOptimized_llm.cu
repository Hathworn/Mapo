#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define size 1024
#define block_size 32

__global__ void matrixMulOptimized(int* a, int* b, int* c)
{
    __shared__ float a_share[block_size][block_size];
    __shared__ float b_share[block_size][block_size];

    int n = size;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int local_c = 0;

    // Iterate through tiles
    for (int i = 0; i < n / block_size; ++i)
    {
        // Load from global memory to shared memory
        a_share[threadIdx.y][threadIdx.x] = a[row * n + i * block_size + threadIdx.x];
        b_share[threadIdx.y][threadIdx.x] = b[(i * block_size + threadIdx.y) * n + col];

        __syncthreads();

        // Compute partial product for this tile
        #pragma unroll
        for (int k = 0; k < block_size; ++k)
        {
            local_c += a_share[threadIdx.y][k] * b_share[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Write back the result to global memory
    c[row * n + col] = local_c;
}