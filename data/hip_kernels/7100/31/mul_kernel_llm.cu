#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify grid and block computation
    int gridStride = blockDim.x * gridDim.x; // Calculate the total stride

    for (; i < N; i += gridStride) // Use loop to handle data larger than a block
    {
        Y[i*INCY] *= X[i*INCX]; // Perform operation
    }
}