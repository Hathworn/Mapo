#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    float sum = 0;

    // Loop unrolling for better performance
    for (int j = 0; j < batch; ++j){
        int index = j * spatial * filters + filter * spatial + id;
        for (int i = 0; i < spatial; i += threads) {
            if (i + id < spatial)
                sum += delta[index + i];
        }
    }
    local[id] = sum;

    __syncthreads();  // Ensure all additions are complete

    // Use reduction to sum in shared memory
    if (threads >= 1024) { if (id < 512) local[id] += local[id + 512]; __syncthreads(); }
    if (threads >= 512) { if (id < 256) local[id] += local[id + 256]; __syncthreads(); }
    if (threads >= 256) { if (id < 128) local[id] += local[id + 128]; __syncthreads(); }
    if (threads >= 128) { if (id < 64) local[id] += local[id + 64]; __syncthreads(); }
    
    // Final reduction
    if (id < 32) {
        volatile float* vlocal = local;
        vlocal[id] += vlocal[id + 32];
        vlocal[id] += vlocal[id + 16];
        vlocal[id] += vlocal[id + 8];
        vlocal[id] += vlocal[id + 4];
        vlocal[id] += vlocal[id + 2];
        vlocal[id] += vlocal[id + 1];
    }

    if (id == 0) {
        mean_delta[filter] = local[0] * (-1. / sqrt(variance[filter] + .000001f));
    }
}