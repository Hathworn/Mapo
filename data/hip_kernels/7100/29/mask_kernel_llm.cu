#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified calculation of 'i'

    if (i < n && mask[i] == mask_num) // Kept conditional check
    {
        x[i] = mask_num;
    }
}