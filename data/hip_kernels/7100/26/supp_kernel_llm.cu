#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize to avoid redundant calculation of (X[i*INCX] * X[i*INCX])
    if (i < N) {
        float value = X[i * INCX];
        if ((value * value) < (ALPHA * ALPHA)) {
            X[i * INCX] = 0;
        }
    }
}