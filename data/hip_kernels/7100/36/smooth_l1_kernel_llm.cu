#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Use a single index calculation to improve readability and performance
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x; 
    if(i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff); // Use intrinsic for better performance
        if(abs_val < 1.0f) {
            error[i] = diff * diff;
            delta[i] = diff;
        } else {
            error[i] = 2.0f * abs_val - 1.0f;
            delta[i] = copysignf(1.0f, diff); // Use intrinsic for branch reduction
        }
    }
}