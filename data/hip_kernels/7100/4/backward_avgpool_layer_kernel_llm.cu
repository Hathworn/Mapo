#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel using shared memory to reduce repetitive calculations
__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    int b = id / c;  // Simplified division

    int out_index = k + c * b;
    float out_value = out_delta[out_index] / (w * h);  // Pre-compute division

    int base_in_index = h * w * (k + b * c);
    for(int i = 0; i < w * h; ++i) {
        int in_index = i + base_in_index;
        in_delta[in_index] += out_value;
    }
}