#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Optimize loop to process multiple elements per thread
    for (; i < n; i += stride) {
        c[i] += a[i] * b[i];
    }
}