#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    int filter = blockIdx.y; // Cache bias index
    float bias = biases[filter];

    // Use a single loop to calculate the full index directly
    for(int offset = blockIdx.x * blockDim.x + threadIdx.x;
        offset < size;
        offset += blockDim.x * gridDim.x)
    {
        int index = (blockIdx.z * n + filter) * size + offset;
        output[index] += bias; // Apply bias within bounds
    }
}