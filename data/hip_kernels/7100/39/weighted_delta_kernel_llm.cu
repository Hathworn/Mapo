#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate global index for the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    if (i < n) {
        float dc_i = dc[i];
        float s_i = s[i];
        
        // Conditional update of da[i]
        if (da) da[i] += dc_i * s_i;
        
        // Update db[i] using local variables
        db[i] += dc_i * (1 - s_i);

        // Update ds[i] with atomicAdd to prevent race conditions
        atomicAdd(&ds[i], dc_i * (a[i] - b[i]));
    }
}