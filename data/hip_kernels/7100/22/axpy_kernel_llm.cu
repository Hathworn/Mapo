#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate the global thread index using only blockIdx.x
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a while loop to handle multiple iterations per thread
    while (i < N) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
        i += blockDim.x * gridDim.x; // Move to the next iteration
    }
}