#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    // Calculate the global index
    int index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Only process valid indices
    if (index < N) {
        // Compute the filter index
        int f = (index / spatial) % filters;
        
        // Pre-calculate constant values to avoid redundant computations
        float inv_std = 1.0f / (sqrtf(variance[f]) + 1e-6f);
        float coeff = variance_delta[f] * 2.0f / (spatial * batch);
        
        // Update delta
        delta[index] = delta[index] * inv_std +
                       coeff * (x[index] - mean[f]) +
                       mean_delta[f] / (spatial * batch);
    }
}