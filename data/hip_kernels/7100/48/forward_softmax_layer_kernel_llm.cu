#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define WARP_SIZE 32

__global__ void forward_softmax_layer_kernel(int n, int batch, float *input, float temp, float *output)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if (b >= batch) return;

    float sum = 0;
    float largest = -INFINITY;

    // Compute largest value in warp for numerical stability
    for (int i = threadIdx.x; i < n; i += WARP_SIZE) {
        float val = input[i + b * n];
        largest = max(largest, val);
    }

    largest = warpReduceMax(largest); // Efficient max reduction using warp functions

    // Compute softmax sum
    for (int i = threadIdx.x; i < n; i += WARP_SIZE) {
        sum += exp(input[i + b * n] / temp - largest / temp);
    }

    sum = warpReduceSum(sum); // Efficient sum reduction using warp functions

    sum = max(sum, INFINITY); // Ensuring sum is never zero

    float offset = largest / temp + log(sum);

    // Calculate softmax output
    for (int i = threadIdx.x; i < n; i += WARP_SIZE) {
        output[i + b * n] = exp(input[i + b * n] / temp - offset);
    }
}

// Warp reduction functions
__device__ float warpReduceMax(float val) {
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        val = max(val, __shfl_down(val, offset));
    }
    return val;
}

__device__ float warpReduceSum(float val) {
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        val += __shfl_down(val, offset);
    }
    return val;
}
```
