#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Use blockIdx.x, blockDim.x for better understanding and maintainability
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = gridDim.x * blockDim.x;
    
    // Loop to allow handling larger data by each thread
    for(int idx = i; idx < N; idx += gridStride)
    {
        Y[idx * INCY] = powf(X[idx * INCX], ALPHA); // Use powf for single precision
    }
}