#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = id; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i;
            local[id] += x[index]; // Compute sum with stride access
        }
    }

    __syncthreads(); // Synchronize threads in block

    // Reduce local array to compute mean
    if (threads >= 1024) { if (id < 512) { local[id] += local[id + 512]; } __syncthreads(); }
    if (threads >= 512) { if (id < 256) { local[id] += local[id + 256]; } __syncthreads(); }
    if (threads >= 256) { if (id < 128) { local[id] += local[id + 128]; } __syncthreads(); }
    if (threads >= 128) { if (id < 64) { local[id] += local[id + 64]; } __syncthreads(); }
    if (id < 32) {
        local[id] += local[id + 32];
        local[id] += local[id + 16];
        local[id] += local[id + 8];
        local[id] += local[id + 4];
        local[id] += local[id + 2];
        local[id] += local[id + 1];
    }

    if (id == 0){
        mean[filter] = local[0] / (spatial * batch); // Calculate mean
    }
}