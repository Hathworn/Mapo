#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Calculate unique global thread index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if within bounds and parallelize conditionally
    if (i < n) {
        binary[i] = __frsqrt_rn(__frsqrt_rn(x[i]) >= 0 ? 1.0f : -1.0f); // Simplified condition
    }
}