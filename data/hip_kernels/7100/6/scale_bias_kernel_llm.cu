#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    if (offset >= size) return; // Early exit for non-valid threads

    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Use registers for repeated index calculations
    int idx = (batch * n + filter) * size + offset;
    output[idx] *= biases[filter];
}