#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    extern __shared__ float shared_mean[]; // Use shared memory for mean to reduce global memory access
    float scale = 1.0f / (batch * spatial - 1);
    int j, k;
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify thread index calculation
    if (i >= filters) return;

    shared_mean[threadIdx.x] = mean[i]; // Load mean into shared memory
    __syncthreads(); // Ensure all threads have loaded mean

    float var_temp = 0.0f; // Local accumulator for variance
    for (j = 0; j < batch; ++j) {
        for (k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            var_temp += (x[index] - shared_mean[threadIdx.x]) * (x[index] - shared_mean[threadIdx.x]); // Avoid pow for efficiency
        }
    }
    variance[i] = var_temp * scale;
}