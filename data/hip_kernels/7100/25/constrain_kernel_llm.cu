#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index with blockDim.x
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N) {
        // Perform constraining operations on array X
        float val = X[i*INCX];
        if (val > ALPHA) {
            X[i*INCX] = ALPHA;
        } else if (val < -ALPHA) {
            X[i*INCX] = -ALPHA;
        }
    }
}