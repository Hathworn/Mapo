#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int c = in_c;
    int area = (size - 1) / stride;
    
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Calculate b, k, i, j using modulo and divide operations
    int index = id;
    int j = index % in_w;
    index /= in_w;
    int i = index % in_h;
    index /= in_h;
    int k = index % in_c;
    int b = index / in_c;

    int w_offset = -pad;
    int h_offset = -pad;

    float d = 0.0f;

    // Pre-calculate common factors outside of loops
    int base_out_index = k + c * b;

    // Use more efficient boundary calculations in loops
    for (int l = -area; l <= area; ++l) {
        int out_h = (i - h_offset) / stride + l;
        if (out_h < 0 || out_h >= h) continue;  // Skip out-of-bounds heights
        for (int m = -area; m <= area; ++m) {
            int out_w = (j - w_offset) / stride + m;
            if (out_w < 0 || out_w >= w) continue;  // Skip out-of-bounds widths
            int out_index = out_w + w * (out_h + h * base_out_index);
            if (indexes[out_index] == id) {  // Check if index matches and add delta
                d += delta[out_index];
            }
        }
    }
    prev_delta[id] += d;  // Accumulate the result into prev_delta
}