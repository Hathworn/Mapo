#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    // Calculate unique thread index
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;

    // Precompute terms used in multiple calculations
    float B1_t_pow = powf(B1, t);
    float B2_t_pow = powf(B2, t);
    float rate_adj = rate * sqrtf(1.0f - B2_t_pow) / (1.0f - B1_t_pow);

    // Optimize division by using reciprocal
    x[index] -= rate_adj * m[index] / (sqrtf(v[index]) + eps);
}