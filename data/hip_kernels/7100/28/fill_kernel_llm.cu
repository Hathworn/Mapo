#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread ID, optimized for 1D grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Using a stride loop to cover all elements
    for (; i < N; i += blockDim.x * gridDim.x) {
        X[i * INCX] = ALPHA;
    }
}