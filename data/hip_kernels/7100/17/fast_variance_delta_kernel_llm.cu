#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Use a stride loop to optimize memory access pattern
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += delta[index] * (x[index] - mean[filter]);
        }
    }

    __syncthreads(); // ensure all threads have written to local before reduction

    // Parallel reduction within block
    for (int s = threads / 2; s > 0; s >>= 1) {
        if (id < s) {
            local[id] += local[id + s];
        }
        __syncthreads();
    }

    if (id == 0) {
        variance_delta[filter] = local[0] * -.5 * pow(variance[filter] + .000001f, (float)(-3. / 2.));
    }
}