#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCK 256

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Use stride loop to improve memory access
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Reduce within the block using warp-level reduction
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (p < stride) {
            part[p] += part[p + stride];
        }
        __syncthreads();
    }

    if (p == 0) {
        bias_updates[filter] += part[0]; // Accumulate results from each block
    }
}