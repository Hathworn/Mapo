#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    // Calculate the global thread index
    int s = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (s >= size) return;

    float mean = 0;
    // Use shared memory for partial sums
    __shared__ float partial_sum[256]; // Assume blockDim.x <= 256

    // Calculate partial sums of absolute values
    float sum = 0;
    for (int i = 0; i < n; ++i) {
        sum += abs(input[i * size + s]);
    }

    // Store partial sum in shared memory
    partial_sum[threadIdx.x] = sum / n;

    // Reduce shared memory to compute mean
    __syncthreads();
    if (threadIdx.x == 0) {
        float block_mean = 0;
        for (int j = 0; j < blockDim.x; ++j) {
            block_mean += partial_sum[j];
        }
        mean = block_mean / blockDim.x;
    }
    __syncthreads();

    // Binarize the input based on computed mean
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}