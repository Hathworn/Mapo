#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate unique global thread index
    if(idx >= N) return;

    // Calculate indices and addresses more efficiently
    int in_index = idx;
    int in_w = in_index % w;                   // in_w calculation
    in_index /= w;
    int in_h = in_index % h;                   // in_h calculation
    in_index /= h;
    int in_c = in_index % c;                   // in_c calculation
    in_index /= c;
    int b = in_index % batch;                  // batch calculation

    int out_c = c / (stride * stride);         // Calculate output channel

    int c2 = in_c % out_c;                     // Calculate channel offset
    int offset = in_c / out_c;
    int w2 = in_w * stride + (offset % stride); // Calculate output width
    int h2 = in_h * stride + (offset / stride); // Calculate output height

    // Calculate output index
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    // Conditional operator for compactness
    if (forward) {
        out[out_index] = x[idx];
    } else {
        out[idx] = x[out_index];
    }
}