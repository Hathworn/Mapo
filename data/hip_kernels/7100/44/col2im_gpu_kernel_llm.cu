#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void col2im_gpu_kernel(const int n, const float* data_col, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_im) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x; // Precompute total threads

    // Looping through all elements with stride
    for (int i = index; i < n; i += total_threads) {
        float val = 0;
        int w = i % width + pad;
        int h = (i / width) % height + pad;
        int c = i / (width * height);

        // Precompute these values outside the inner loops
        int w_col_start = max(0, (w - ksize) / stride + 1);
        int w_col_end = min(w / stride + 1, width_col);
        int h_col_start = max(0, (h - ksize) / stride + 1);
        int h_col_end = min(h / stride + 1, height_col);
        
        int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col;
        int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
        int coeff_w_col = (1 - stride * height_col * width_col);

        // Use a single loop for simplicity and potential unrolling
        for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
            for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
                int col_index = offset + h_col * coeff_h_col + w_col * coeff_w_col;
                val += data_col[col_index];
            }
        }
        data_im[i] += val;
    }
}