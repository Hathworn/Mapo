#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    int h = (in_h + 2*pad)/stride;
    int w = (in_w + 2*pad)/stride;
    int c = in_c;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Exit early if id is out of bounds
    if(id >= n) return;

    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    int w_offset = -pad;
    int h_offset = -pad;

    int out_index = j + w*(i + h*(k + c*b));
    float max = -INFINITY;
    int max_i = -1;

    // Unroll the nested loop to potentially increase performance
    #pragma unroll
    for(int l = 0; l < size; ++l){
        for(int m = 0; m < size; ++m){
            int cur_h = h_offset + i*stride + l;
            int cur_w = w_offset + j*stride + m;
            int index = cur_w + in_w*(cur_h + in_h*(k + b*in_c));
            int valid = (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w);
            float val = (valid != 0) ? input[index] : -INFINITY;

            if (val > max) {
                max = val;
                max_i = index;
            }
        }
    }

    output[out_index] = max;
    indexes[out_index] = max_i;
}