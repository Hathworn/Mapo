#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float temp_variance_delta = 0.0f; // Local variable to accumulate result
    float mean_val = mean[i];
    float variance_val = variance[i];
    float factor = -.5f * pow(variance_val + .000001f, -1.5f); // Precompute constant factor

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            temp_variance_delta += delta[index] * (x[index] - mean_val);
        }
    }

    variance_delta[i] = temp_variance_delta * factor; // Apply factor after accumulation
}