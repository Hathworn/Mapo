#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    float sum = 0;
    float largest = -INFINITY;
    // Use parallel reduction to find the largest value
    for(int i = 0; i < n; ++i){
        float val = input[i];
        largest = fmaxf(val, largest);
    }
    for(int i = 0; i < n; ++i){
        float e = expf((input[i] - largest) / temp);  // Optimize temp division outside exp
        sum += e;
        output[i] = e;
    }
    // Inverse sum needed
    float sum_inv = 1.0f / sum;
    for(int i = 0; i < n; ++i){
        output[i] *= sum_inv;  // Replace division with multiplication
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify block and thread index calculation
    if(b >= batch) return;
    softmax_device(n, input + b*offset, temp, output + b*offset);
}