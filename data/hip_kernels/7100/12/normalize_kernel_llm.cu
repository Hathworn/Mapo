#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    // Calculate index using faster grid-stride loop
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < N; index += blockDim.x * gridDim.x) {
        int f = (index / spatial) % filters;

        // Perform normalization
        x[index] = (x[index] - mean[f]) / (sqrt(variance[f]) + .000001f);
    }
}