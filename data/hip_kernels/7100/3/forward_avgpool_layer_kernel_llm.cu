#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int k = id % c;
    int b = id / c;

    // Utilize shared memory for temporary storage
    extern __shared__ float shared_input[];
    int out_index = k + c * b;
    shared_input[threadIdx.x] = 0;

    // Coalesce global memory access by aligning memory access patterns
    for (int i = threadIdx.x; i < w * h; i += blockDim.x) {
        int in_index = i + h * w * (k + b * c);
        shared_input[threadIdx.x] += input[in_index];
    }

    // Perform a reduction within the block
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            shared_input[threadIdx.x] += shared_input[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Write result to global memory
    if(threadIdx.x == 0){
        output[out_index] = shared_input[0] / (w * h);
    }
}