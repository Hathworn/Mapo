#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    // Initialize local shared memory
    local[id] = 0.0f;

    // Accumulate variance for each element processed by this thread
    for(int j = 0; j < batch; ++j){
        for(int i = id; i < spatial; i += threads){  // Optimize loop stride
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += pow((x[index] - mean[filter]), 2);
        }
    }

    // Use parallel reduction to sum up local variances
    __syncthreads();
    for(int offset = threads / 2; offset > 0; offset >>= 1) {
        if(id < offset) {
            local[id] += local[id + offset];
        }
        __syncthreads();
    }

    // Write final result to global memory
    if(id == 0) {
        variance[filter] = local[0] / (spatial * batch - 1);
    }
}