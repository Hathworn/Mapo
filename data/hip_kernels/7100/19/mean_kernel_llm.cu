#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Compute the global index for the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Use float atomicAdd to handle concurrent writes safely and potentially optimize memory operations
    float sum = 0.0f;
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += x[index];
        }
    }

    // Calculate scale once per iteration instead of every time
    float scale = 1.0f / (batch * spatial);
    mean[i] = sum * scale;
}