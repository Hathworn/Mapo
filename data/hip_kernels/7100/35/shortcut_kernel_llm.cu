#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride_grid = blockDim.x * gridDim.x;  // Calculate grid stride

    for (; id < size; id += stride_grid) {
        int temp_id = id;
        int i = temp_id % minw;  // Calculate the index for w1 dimension
        temp_id /= minw;
        int j = temp_id % minh;  // Calculate the index for h1 dimension
        temp_id /= minh;
        int k = temp_id % minc;  // Calculate the index for c1 dimension
        temp_id /= minc;
        int b = temp_id % batch; // Calculate the index for batch dimension

        int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));       // Calculate the out index
        int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));       // Calculate the add index
        out[out_index] += add[add_index];  // Update out with add value
    }
}
```
