#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global index using only blockIdx.x
    int stride = gridDim.x * blockDim.x; // Determine the stride for loop unrolling

    for (; i < n; i += stride) { // Loop unrolling: process multiple elements in steps
        float diff = truth[i] - pred[i]; // Calculate difference
        error[i] = diff * diff; // Compute error
        delta[i] = diff; // Store delta
    }
}