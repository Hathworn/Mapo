#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0.0f;

    // Unroll the loop for better performance
    for (int b = 0; b < batch; ++b){
        for (int i = p; i < size; i += BLOCK){
            int index = i + size * (filter + n * b);
            sum += delta[index] * x_norm[index];
        }
    }

    part[p] = sum;
    __syncthreads();

    // Use parallel reduction for better performance
    if (BLOCK >= 1024 && p < 512) { part[p] += part[p + 512]; __syncthreads(); }
    if (BLOCK >= 512 && p < 256) { part[p] += part[p + 256]; __syncthreads(); }
    if (BLOCK >= 256 && p < 128) { part[p] += part[p + 128]; __syncthreads(); }
    if (BLOCK >= 128 && p < 64) { part[p] += part[p + 64]; __syncthreads(); }
    if (p < 32) {
        if (BLOCK >= 64) part[p] += part[p + 32];
        if (BLOCK >= 32) part[p] += part[p + 16];
        if (BLOCK >= 16) part[p] += part[p + 8];
        if (BLOCK >= 8) part[p] += part[p + 4];
        if (BLOCK >= 4) part[p] += part[p + 2];
        if (BLOCK >= 2) part[p] += part[p + 1];
    }

    // Final update to scale_updates
    if (p == 0) {
        scale_updates[filter] += part[0];
    }
}