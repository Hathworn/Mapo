#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    // Calculate thread ID using 3D block and thread indices for better scalability
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = blockIdx.y * gridDim.x * blockDim.x + id;

    // Check within bounds and update input using rand and scale
    if (offset < size) {
        input[offset] = (rand[offset] < prob) ? 0 : input[offset] * scale;
    }
}