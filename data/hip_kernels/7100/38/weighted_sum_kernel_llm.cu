#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate the unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within bounds and coalesce b's optional handling
    if (i < n)
    {
        float b_value = b ? b[i] : 0.0f;
        c[i] = s[i] * a[i] + (1 - s[i]) * b_value;
    }
}