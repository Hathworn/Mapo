#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use shared memory to potentially reduce global memory accesses
    int idx = blockIdx.x * blockDim.x + threadIdx.x;       // Calculate the global thread index
    int stride = gridDim.x * blockDim.x;                   // Calculate the grid stride

    // Loop over the input data with grid stride
    for (int i = idx; i < N; i += stride)
    {
        X[i * INCX] = ALPHA;
    }
}