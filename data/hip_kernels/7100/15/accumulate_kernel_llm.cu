#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= groups) return;
    
    float local_sum = 0; // Declare local variable for sum
    
    for(int k = 0; k < n; ++k){
        local_sum += x[k * groups + i]; // Accumulate in local variable to reduce global memory access
    }
    
    sum[i] = local_sum; // Write result back to global memory once
}