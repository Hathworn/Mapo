#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Optimize thread indexing calculation
    if(i >= N) return;

    int b = i / (layers * spatial);  // Precompute b to reduce redundant calculations
    int in_c = (i / spatial) % layers;  // Optimize the index calculations using properties of modulo and division
    int in_s = i % spatial;

    int i1 = b * layers * spatial + in_c * spatial + in_s;  // Optimize memory access pattern
    int i2 = b * layers * spatial + in_s * layers + in_c;

    if (forward) 
        out[i2] = x[i1];  // Optimize conditional operation
    else 
        out[i1] = x[i2];
}