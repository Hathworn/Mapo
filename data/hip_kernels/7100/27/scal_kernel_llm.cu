#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Launch multiple blocks along y-axis to cover entire data in case of large N
    int stride = blockDim.x * gridDim.x;

    // Use a loop to process multiple elements per thread to ensure full GPU utilization
    for (int idx = i; idx < N; idx += stride) {
        X[idx * INCX] *= ALPHA;
    }
}