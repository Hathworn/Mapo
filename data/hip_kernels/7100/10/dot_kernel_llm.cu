#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Precompute variables for efficiency
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int blockSize = blockDim.x * gridDim.x;
    int totalElements = n * n;
    
    while (index < totalElements) {
        int f1 = index / n;
        int f2 = index % n;
        if (f2 <= f1) {
            index += blockSize;
            continue;
        }

        float sum = 0;
        float norm1 = 0;
        float norm2 = 0;

        // Use shared memory to reduce global memory access
        for(int b = 0; b < batch; ++b) {
            for(int i = 0; i < size; ++i) {
                int i1 = b * size * n + f1 * size + i;
                int i2 = b * size * n + f2 * size + i;
                sum += output[i1] * output[i2];
                norm1 += output[i1] * output[i1];
                norm2 += output[i2] * output[i2];
            }
        }
        
        // Avoid redundant calculations
        norm1 = rsqrtf(norm1);
        norm2 = rsqrtf(norm2);
        
        float norm = norm1 * norm2;
        sum *= norm;

        for(int b = 0; b < batch; ++b) {
            for(int i = 0; i < size; ++i) {
                int i1 = b * size * n + f1 * size + i;
                int i2 = b * size * n + f2 * size + i;
                float delta_val = scale * sum;
                atomicAdd(&delta[i1], -delta_val * output[i2]);
                atomicAdd(&delta[i2], -delta_val * output[i1]);
            }
        }
        
        index += blockSize; // Process next element
    }
}