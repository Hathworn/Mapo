#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil_1d(int *in, int *out) {
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];

    // Calculate global and local index for current thread
    int gindex = threadIdx.x + (blockIdx.x * blockDim.x) + RADIUS;
    int lindex = threadIdx.x + RADIUS;

    // Load current element and halo into shared memory
    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS) {
        temp[lindex - RADIUS] = in[gindex - RADIUS];
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }

    // Synchronize threads to ensure shared memory is fully populated
    __syncthreads();

    // Compute the result using stencil pattern
    int result = 0;
    #pragma unroll // Unroll loop to optimize performance
    for (int offset = -RADIUS; offset <= RADIUS; offset++) {
        result += temp[lindex + offset];
    }

    // Store the computed result in global memory
    out[gindex - RADIUS] = result;
}