#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Calculate the global index for this thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform addition for this specific index
    c[idx] = a[idx] + b[idx];
}