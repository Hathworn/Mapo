#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int* c, const int* a, const int* b, int size) {
    // Calculate global index for current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread does not process beyond array size
    if (i < size) {
        c[i] = a[i] + b[i]; // Perform vector addition
    }
}