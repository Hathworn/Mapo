#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x; // Calculate global index
    c[index] = a[index] + b[index]; // Use global index
}