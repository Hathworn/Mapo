#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use a grid-stride loop to ensure all threads can execute printf if there are more threads than work items
__global__ void hello_kernel(void)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = idx; i < 1; i += stride) {
        printf("Hello, world from the device!\n");
    }
}