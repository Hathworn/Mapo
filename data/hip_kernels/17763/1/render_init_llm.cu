#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    // Use shared memory for faster access in cache
    extern __shared__ hiprandState shared_rand_state[];
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if((i >= max_x) || (j >= max_y)) return;

    int pixel_index = j * max_x + i;

    // Each thread initializes its own state in shared memory
    hiprand_init(1984, pixel_index, 0, &shared_rand_state[threadIdx.x + threadIdx.y * blockDim.x]);

    // Copy shared state back to global memory
    rand_state[pixel_index] = shared_rand_state[threadIdx.x + threadIdx.y * blockDim.x];
}