#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c, int n)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index is within array bounds
    if (index >= n) return;

    // Perform the addition
    c[index] = a[index] + b[index];
}