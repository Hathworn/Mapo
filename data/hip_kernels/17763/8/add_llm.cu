#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Using shared memory to reduce global memory access
    __shared__ int shared_a, shared_b;
    
    // Load the values into shared memory per block
    shared_a = *a;
    shared_b = *b;
    
    // Ensure all threads are synchronized before proceeding
    __syncthreads();

    // Perform the addition using shared memory variables
    *c = shared_a + shared_b;
    printf("Result %d ", *c);
}