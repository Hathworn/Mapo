#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/* ==================================================================
Programmers:
Kevin Wagner
Elijah Malaby
John Casey

Omptimizing SDH histograms for input larger then global memory
==================================================================
*/

#define BOX_SIZE 23000 /* size of the data box on one dimension */

/* descriptors for single atom in the tree */
typedef struct atomdesc {
    float x_pos;
    float y_pos;
    float z_pos;
} atom;

unsigned long long * histogram;	/* list of all buckets in the histogram */
unsigned long long  PDH_acnt;	/* total number of data points */
int block_size;	/* Number of threads per block */
int num_buckets;	/* total number of buckets in the histogram */
float PDH_res;	/* value of w */
atom * atom_list;	/* list of all data points */
unsigned long long * histogram_GPU;
unsigned long long * temp_histogram_GPU;
atom * atom_list_GPU;

__device__ void block_to_block (atom * block_a, atom * block_b, int b_length, unsigned long long * histogram, float resolution) {
    atom me = block_a[threadIdx.x];
    for (int i = 0; i < b_length; i++) {
        float dx = me.x_pos - block_b[i].x_pos;
        float dy = me.y_pos - block_b[i].y_pos;
        float dz = me.z_pos - block_b[i].z_pos;
        int bin = (int)(sqrt(dx * dx + dy * dy + dz * dz) / resolution);
        atomicAdd(&(histogram[bin]), 1);
    }
}

__global__ void GPUKernelFunction (unsigned long long PDH_acnt, float PDH_res, atom * atom_list_GPU, unsigned long long * histogram_GPU, int num_buckets) {
    extern __shared__ unsigned long long SHist[];
    // initialize shared memory histogram
    for (int h_pos = threadIdx.x; h_pos < num_buckets; h_pos += blockDim.x) {
        SHist[h_pos] = 0;
    }
    __syncthreads();

    // get atoms for the current block
    atom * my_block = &atom_list_GPU[blockIdx.x * blockDim.x];
    atom temp_atom_1 = my_block[threadIdx.x];

    // compute distances within the block
    for (int i = threadIdx.x + 1; i < blockDim.x && i + blockIdx.x * blockDim.x < PDH_acnt; i++) {
        atom temp_atom_2 = my_block[i];
        float dx = temp_atom_1.x_pos - temp_atom_2.x_pos;
        float dy = temp_atom_1.y_pos - temp_atom_2.y_pos;
        float dz = temp_atom_1.z_pos - temp_atom_2.z_pos;
        int h_pos = (int)(sqrt(dx * dx + dy * dy + dz * dz) / PDH_res);
        atomicAdd(&(SHist[h_pos]), 1);
    }
    __syncthreads();

    // compute distances between different blocks
    for (int i = blockIdx.x + 1; i < gridDim.x - 1; i++) {
        block_to_block(my_block, &atom_list_GPU[i * blockDim.x], blockDim.x, SHist, PDH_res);
    }
    block_to_block(my_block, &atom_list_GPU[i * blockDim.x], PDH_acnt - i * blockDim.x, SHist, PDH_res); // Last block may be small

    __syncthreads();

    // update global histogram
    for (int h_pos = threadIdx.x; h_pos < num_buckets; h_pos += blockDim.x) {
        atomicAdd(histogram_GPU + blockIdx.x * num_buckets + h_pos, SHist[h_pos]);
    }
}
```
