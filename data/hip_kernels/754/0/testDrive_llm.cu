#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int test()
{
    return 10;
}

__global__ void testDrive()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use warp shuffle to reduce redundant calculations
    int lane = index % warpSize;
    int num;
    
    if (lane == 0) {
        num = test();
    }
    
    num = __shfl_sync(0xFFFFFFFF, num, 0);
    
    printf("index = %d, num = %d\n", index, num);
}