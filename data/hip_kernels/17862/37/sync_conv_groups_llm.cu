#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups() {
    // Simplified and optimized for readability
    __syncthreads(); // Ensure block-level synchronization
}