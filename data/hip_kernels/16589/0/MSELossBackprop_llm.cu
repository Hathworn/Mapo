#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

/*** Definitions ***/
// Block width for CUDA kernels
#define BW 128
#define RANDOM_SEED -1

#ifdef USE_GFLAGS

#ifndef _WIN32
#define gflags google
#endif
#else
// Constant versions of gflags
#define DEFINE_int32(flag, default_value, description) const int FLAGS_##flag = (default_value)
#define DEFINE_uint64(flag, default_value, description) const unsigned long long FLAGS_##flag = (default_value)
#define DEFINE_bool(flag, default_value, description) const bool FLAGS_##flag = (default_value)
#define DEFINE_double(flag, default_value, description) const double FLAGS_##flag = (default_value)
#define DEFINE_string(flag, default_value, description) const std::string FLAGS_##flag ((default_value))
#endif

__global__ void MSELossBackprop(float *grad_data, float *output, float *target, float *mask, int batch_size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size)
        return;

    // Pre-calculate value difference
    float diff = output[idx] - target[idx];

    // Use switch statement for better readability and efficiency
    switch (__float2int_rd(mask[idx])) {
        case -1:
            grad_data[idx] = 0.05f * diff;
            break;
        case 1:
            grad_data[idx] = 5.0f * diff;
            break;
        default:
            grad_data[idx] = 0.0f;
            break;
    }
}