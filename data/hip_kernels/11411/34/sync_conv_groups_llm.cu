#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function for synchronization across conv groups
__global__ void sync_conv_groups() {
    // Using __syncthreads to ensure all threads have reached this point
    __syncthreads();
}