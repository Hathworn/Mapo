#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    // Calculate the output dimensions
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int c = in_c;
    int area = (size - 1) / stride;

    // Thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int index = id;
    int j = index % in_w;
    index /= in_w;
    int i = index % in_h;
    index /= in_h;
    int k = index % in_c;
    index /= in_c;
    int b = index;

    int w_offset = -pad;
    int h_offset = -pad;

    // Initialize delta accumulator
    float d = 0;

    // Use loop unrolling for small 'area' ranges
    for (int l = -area; l <= area; ++l) {
        for (int m = -area; m <= area; ++m) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;
            int out_index = out_w + w * (out_h + h * (k + c * b));
            bool valid = (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h);

            // Leverage boolean short-circuit and skip unnecessary operations
            if (valid && indexes[out_index] == id) {
                d += delta[out_index];
            }
        }
    }

    // Accumulate the calculated delta
    prev_delta[id] += d;
}