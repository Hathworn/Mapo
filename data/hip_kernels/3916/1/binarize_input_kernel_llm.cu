#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    // Calculate global thread index
    int s = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    // Check boundary condition
    if (s >= size) return;

    // Optimize mean calculation with shared memory
    extern __shared__ float shared_mem[];
    float* shared_mean = shared_mem;

    int i = threadIdx.x;
    if (i < n) {
        shared_mean[i] = fabsf(input[i * size + s]);
    }
    __syncthreads();

    // Reduce to compute mean
    if (i == 0) {
        float mean = 0;
        for (int j = 0; j < n; ++j) {
            mean += shared_mean[j];
        }
        mean = mean / n;

        // Update binary results
        for (int j = 0; j < n; ++j) {
            binary[j * size + s] = (input[j * size + s] > 0) ? mean : -mean;
        }
    }
}