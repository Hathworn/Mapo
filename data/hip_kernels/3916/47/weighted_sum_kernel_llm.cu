#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid-stride loop for better utilization of GPU resources
    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x) {
        c[idx] = s[idx] * a[idx] + (1.0f - s[idx]) * (b ? b[idx] : 0.0f);
    }
}