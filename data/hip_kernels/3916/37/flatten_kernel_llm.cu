#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Compute global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (index >= N) return;

    // Calculate in_s, in_c, and batch index b
    int in_s = index % spatial;
    int layer_index = index / spatial;
    int in_c = layer_index % layers;
    int b = layer_index / layers;

    // Calculate flat index for input and output
    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;

    // Assign values based on forwarding condition
    if (forward) out[i2] = x[i1];
    else out[i1] = x[i2];
}