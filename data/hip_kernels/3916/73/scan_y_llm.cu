#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan_y(int *g_odata, int *g_idata, int n)
{
    extern __shared__ int temp[];  // allocated on invocation
    int thid = threadIdx.x;
    int bid = blockIdx.x;
    int bdim = blockDim.x;
    int gdim = gridDim.x;
    int offset = 1;

    // Use cooperative loading to maximize shared memory usage
    int ai = bid + (2 * thid) * gdim;
    int bi = ai + gdim;
    temp[2 * thid] = g_idata[ai]; 
    temp[2 * thid + 1] = g_idata[bi];

    // Build the sum in place up the tree
    for (int d = n >> 1; d > 0; d >>= 1) {
        __syncthreads();
        if (thid < d) {
            int indexA = offset * (2 * thid + 1) - 1;
            int indexB = offset * (2 * thid + 2) - 1;
            temp[indexB] += temp[indexA];
        }
        offset <<= 1;
    }

    // Clear the last element for downward pass
    if (thid == 0) {
        temp[n - 1] = 0;
    }

    // Traverse down tree and build scan
    for (int d = 1; d < n; d <<= 1) {
        offset >>= 1;
        __syncthreads();
        if (thid < d) {
            int indexA = offset * (2 * thid + 1) - 1;
            int indexB = offset * (2 * thid + 2) - 1;
            int t = temp[indexA];
            temp[indexA] = temp[indexB];
            temp[indexB] += t;
        }
    }

    __syncthreads();
    // Store results to global memory with improved index calculation
    g_odata[ai] = temp[2 * thid + 1];

    // Use ternary for clearer logic
    g_odata[bi] = (2 * thid + 2 == bdim * 2) ? temp[2 * thid + 1] + g_idata[bi] : temp[2 * thid + 2];
}