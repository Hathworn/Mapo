#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void make_pillar_index_kernel(int* dev_pillar_count_histo, 
                                         int* dev_counter, 
                                         int* dev_pillar_count, 
                                         int* dev_x_coors, 
                                         int* dev_y_coors, 
                                         float* dev_x_coors_for_sub, 
                                         float* dev_y_coors_for_sub, 
                                         float* dev_num_points_per_pillar, 
                                         int* dev_sparse_pillar_map, 
                                         const int max_pillars, 
                                         const int max_points_per_pillar, 
                                         const int GRID_X_SIZE, 
                                         const float PILLAR_X_SIZE, 
                                         const float PILLAR_Y_SIZE, 
                                         const int NUM_INDS_FOR_SCAN)
{
    int x = blockIdx.x;
    int y = threadIdx.x;

    // Load the number of points at this pillar from global to register for faster access
    int num_points_at_this_pillar = dev_pillar_count_histo[y * GRID_X_SIZE + x];

    // Early exit if no points
    if (num_points_at_this_pillar == 0) return;

    // Use atomic add to determine unique index for this thread
    int count = atomicAdd(dev_counter, 1);

    // Check if this count is within the limit
    if (count < max_pillars)
    {
        // Atomically increment pillar count safely
        atomicAdd(dev_pillar_count, 1);

        // Limit the number of points to max_points_per_pillar
        dev_num_points_per_pillar[count] = min(num_points_at_this_pillar, max_points_per_pillar);

        // Store coordinates in respective arrays
        dev_x_coors[count] = x;
        dev_y_coors[count] = y;

        dev_x_coors_for_sub[count] = x * PILLAR_X_SIZE + 0.1f;
        dev_y_coors_for_sub[count] = y * PILLAR_Y_SIZE - 39.9f;

        // Update sparse pillar map
        dev_sparse_pillar_map[y * NUM_INDS_FOR_SCAN + x] = 1;
    }
}