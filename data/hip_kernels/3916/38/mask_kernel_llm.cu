#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Check boundary condition and mask value
    if (i < n)
    {
        float mask_val = mask[i];  // Avoid redundant global memory access
        if (mask_val == mask_num)
        {
            x[i] = val;  // Update value
        }
    }
}