#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Calculate the index for the current thread
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Store inverse variance pre-calculation
    float inv_var = -.5f * powf(variance[i] + .00001f, -1.5f);
    
    // Accumulate variance delta
    float acc = 0.0f;
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            acc += delta[index] * (x[index] - mean[i]);
        }
    }
    // Store the result using pre-calculated inverse variance
    variance_delta[i] = acc * inv_var;
}