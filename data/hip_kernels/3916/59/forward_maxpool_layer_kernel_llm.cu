#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int c = in_c;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -pad;
    int h_offset = -pad;

    int out_index = j + w * (i + h * (k + c * b));
    float max = -INFINITY;
    int max_i = -1;
    int cur_h, cur_w, index;
    float val;
    
    // Optimize: Avoid nested loops where possible
    for (int l = 0; l < size; ++l) {
        cur_h = h_offset + i * stride + l;
        if (cur_h < 0 || cur_h >= in_h) continue; // Skip invalid heights
        for (int m = 0; m < size; ++m) {
            cur_w = w_offset + j * stride + m;
            if (cur_w < 0 || cur_w >= in_w) continue; // Skip invalid widths
            index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            val = input[index];
            if (val > max) {
                max = val;
                max_i = index;
            }
        }
    }
    output[out_index] = max;
    indexes[out_index] = max_i;
}