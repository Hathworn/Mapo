#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Improve memory coalescing
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int idx = i; idx < n; idx += stride) {
        c[idx] += a[idx] * b[idx];
    }
}