#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = blockDim.x * gridDim.x; // Calculate total stride for grid

    for (int i = index; i < N; i += gridStride) { // Loop over grid with stride
        int f = (i / spatial) % filters;
        x[i] = (x[i] - mean[f]) / (sqrtf(variance[f] + .00001f));
    }
}