#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index for the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;  // Calculate stride for efficient grid-stride looping

    // Use grid-stride loop to handle larger arrays efficiently
    for (; i < N; i += stride) {
        X[i * INCX] = ALPHA;
    }
}