#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    if (offset >= size) return; // Early exit if offset is out of bounds
    
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Use of register variable to reduce redundant calculations
    float bias = biases[filter];
    
    // Calculate and update output in a single step
    int index = (batch * n + filter) * size + offset;
    output[index] *= bias;
}