#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = blockDim.x;  // Use blockDim.x for portability
    __shared__ float local[BLOCK];  // Allocate for max BLOCK size
    
    int id = threadIdx.x;
    local[id] = 0.0f;
    
    int filter = blockIdx.x;
    
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {  // Increment by threads
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += (i < spatial) ? delta[index] * (x[index] - mean[filter]) : 0.0f;
        }
    }
    
    __syncthreads();
    
    if (id == 0) {
        float sum = 0.0f;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];  // Accumulate in a local variable
        }
        variance_delta[filter] = sum * -0.5f * powf(variance[filter] + 0.00001f, -1.5f);
    }
}