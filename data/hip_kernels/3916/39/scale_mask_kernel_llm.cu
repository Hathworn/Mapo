#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) { // Optimize for loop by processing multiple strides
        if (mask[i] == mask_num) {
            x[i] *= scale; // Simplify logic with in-loop scaling
        }
    }
}