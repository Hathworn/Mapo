#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;

    // Unroll the batch loop for better performance
    for (int b = 0; b < batch; b += 2) {
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            float output1 = output[i1];
            float output2 = output[i2];

            sum += output1 * output2;
            norm1 += output1 * output1;
            norm2 += output2 * output2;

            if (b + 1 < batch) { // Second loop iteration manually unrolled
                i1 += size * n;
                i2 += size * n;
                output1 = output[i1];
                output2 = output[i2];

                sum += output1 * output2;
                norm1 += output1 * output1;
                norm2 += output2 * output2;
            }
        }
    }

    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    sum = sum / norm;

    // Combine and unroll the second set of loops
    for (int b = 0; b < batch; b += 2) {
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            float output1 = output[i1];
            float output2 = output[i2];

            delta[i1] += -scale * sum * output2 / norm;
            delta[i2] += -scale * sum * output1 / norm;

            if (b + 1 < batch) { // Second loop iteration manually unrolled
                i1 += size * n;
                i2 += size * n;
                output1 = output[i1];
                output2 = output[i2];

                delta[i1] += -scale * sum * output2 / norm;
                delta[i2] += -scale * sum * output1 / norm;
            }
        }
    }
}