#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
    
    // Use shared memory for improved performance
    __shared__ float shared_mean;
    float local_sum = 0;

    // Calculate mean using shared memory
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        local_sum += fabsf(weights[f * size + i]);
    }

    // Reduce sum across threads
    atomicAdd(&shared_mean, local_sum);

    // Synchronize threads to ensure mean calculation is complete
    __syncthreads();

    if (threadIdx.x == 0) {
        shared_mean = shared_mean / size;
    }

    __syncthreads();

    // Binarize weights using the calculated mean
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? shared_mean : -shared_mean;
    }
}