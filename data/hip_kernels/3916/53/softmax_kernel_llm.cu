#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output) 
{
    int i;
    float sum = 0;
    float largest = -INFINITY;

    // Use a single pass to find the largest value
    for (i = 0; i < n; ++i) {
        float val = input[i * stride];  // Ensure float type for `val`
        if (val > largest) largest = val;
    }

    for (i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp);  // Simplify exponent calculation
        sum += e;
        output[i * stride] = e;
    }

    sum = 1.0f / sum;  // Calculate reciprocal once
    for (i = 0; i < n; ++i) {
        output[i * stride] *= sum;  // Multiply once
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output) 
{
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;  // Simplify id calculation
    if (id >= batch * groups) return;

    int b = id / groups;
    int g = id % groups;

    // Call optimized device function
    softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}