#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index using 3D grid and block dimensions directly
    int i = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x; 

    // Ensure to update only the elements within bounds
    if (i < N / INCX) {
        X[i * INCX] += ALPHA;
    }
}