#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread index is within bounds
    if(i < n) {
        float t = truth[i];
        float p = pred[i];
        
        // Use a temporary variable to avoid redundant memory accesses.
        float err = 0.0;
        if (t > 0) { // Check for non-zero truth value before error computation
            err = -log(p);
        }
        
        error[i] = err;
        delta[i] = t - p;
    }
}