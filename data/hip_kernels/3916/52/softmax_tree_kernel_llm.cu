#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    float sum = 0;
    float largest = -INFINITY;

    // Unroll the loop for improved performance
    for(int i = 0; i < n; ++i){
        float val = input[i*stride];
        largest = fmaxf(val, largest);
    }
    for(int i = 0; i < n; ++i){
        float e = expf((input[i*stride] - largest) / temp);
        sum += e;
        output[i*stride] = e;
    }
    for(int i = 0; i < n; ++i){
        output[i*stride] /= sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit for threads outside workload
    if (id >= spatial * batch * groups) return;

    int s = id % spatial;
    int group_id = id / spatial;
    int g = group_id % groups;
    int b = group_id / groups;

    int goff = group_offset[g] * spatial;
    int boff = b * stride;

    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}