#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate the global index for each thread
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Use the index to update the Y array, if within bounds
    if (i < N) 
    {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}