#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    // Calculate linear global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    // Precompute (1 - powf(B1, t)) and (1 - powf(B2, t)) for performance improvement
    float invB1t = 1.f - powf(B1, t);
    float invB2t = 1.f - powf(B2, t);

    // Use the precomputed values
    float mhat = m[index] / invB1t;
    float vhat = v[index] / invB2t;

    // Optimize the update step
    x[index] += rate * mhat / (sqrtf(vhat) + eps);
}