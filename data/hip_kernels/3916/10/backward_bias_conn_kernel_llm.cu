#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    // Calculate the unique thread index.
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if index is out of bounds.
    if (index >= n) return;
    
    float sum = 0; // Use register for sum
    
    // Use stride loop for better memory access pattern.
    for (int b = 0; b < batch; ++b) {
        sum += delta[b * n + index];
    }

    // Atomic add to prevent race conditions.
    atomicAdd(&bias_updates[index], sum);
}