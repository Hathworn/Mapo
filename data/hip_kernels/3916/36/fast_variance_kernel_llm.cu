#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[BLOCK];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    int step = batch * spatial * filters;

    local[id] = 0;

    // Use loop unrolling to increase efficiency
    for (int j = 0; j < batch; ++j) {
        int baseIdx = j * spatial * filters + filter * spatial + id;
        int maxIdx = baseIdx + spatial - id;

        // Loop through spatial dimension with stride BLOCK to leverage shared memory
        for (int i = baseIdx; i < maxIdx; i += threads) {
            local[id] += powf((x[i] - mean[filter]), 2);
        }
    }

    __syncthreads();

    // Use atomic operation to reduce synchronization overhead
    if (id == 0) {
        float sum = 0;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        atomicAdd(&variance[filter], sum / (spatial * batch - 1));
    }
}