#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    // Combine blockIdx dimensions and threadIdx to get a unique thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    int b = id / c;

    int out_index = k + c * b;
    float scale = out_delta[out_index] / (w * h);

    // Use a single index variable for the loop to reduce computations inside the loop
    for(int i = 0; i < w * h; ++i){
        int in_index = i + h * w * (k + b * c);
        in_delta[in_index] += scale; // Pre-compute scale to optimize division
    }
}