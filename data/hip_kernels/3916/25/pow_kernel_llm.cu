#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized pow_kernel with improved memory access and parallelization
__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Use 1D grid and block for better mapping
    int stride = gridDim.x * blockDim.x;  // Calculate stride for coalesced memory access
    for (int i = idx; i < N; i += stride) {  // Loop through elements using stride
        Y[i * INCY] = powf(X[i * INCX], ALPHA);  // Use powf for single precision floats
    }
}