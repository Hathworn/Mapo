#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we do not access out-of-bounds memory
    if (index < N) {
        int f = (index / spatial) % filters;

        // Pre-compute frequently used values to reduce redundant calculations
        float inv_sqrt_variance = 1.f / sqrtf(variance[f] + 0.00001f);
        float spatial_batch_inv = 1.f / (spatial * batch);
        float two_variance_delta = 2.f * variance_delta[f];

        float x_mean_diff = x[index] - mean[f];

        // Simplify expression by aggregating similar calculations
        delta[index] = delta[index] * inv_sqrt_variance + 
                       two_variance_delta * x_mean_diff * spatial_batch_inv + 
                       mean_delta[f] * spatial_batch_inv;
    }
}