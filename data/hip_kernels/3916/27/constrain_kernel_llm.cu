#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Flatten block index
    int stride = gridDim.x * blockDim.x;             // Calculate stride for grid loop

    for (int i = idx; i < N; i += stride) {          // Grid stride loop for efficiency
        X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i * INCX])); // Constrain value
    }
}