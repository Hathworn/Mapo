#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize by calculating offsets outside inner loop
__global__ void scatter_kernel(int *x_coors, int *y_coors, float *pfe_output, float *scattered_feature, const int MAX_NUM_PILLARS_, const int GRID_X_SIZE, const int GRID_Y_SIZE)
{
    int i_pillar = blockIdx.x;
    int i_feature = threadIdx.x;

    // Calculate index only once outside of accessing array
    int pillar_idx = i_feature * MAX_NUM_PILLARS_ + i_pillar;
    int scatter_idx = i_feature * GRID_Y_SIZE * GRID_X_SIZE + y_coors[i_pillar] * GRID_X_SIZE + x_coors[i_pillar];

    // Fetch feature and store to global memory
    scattered_feature[scatter_idx] = pfe_output[pillar_idx];
}