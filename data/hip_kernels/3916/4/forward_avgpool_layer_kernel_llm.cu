#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n * c) return; // Ensure all indices are processed

    int k = id % c;
    int b = id / c;

    float sum = 0.0f; // Use a local variable to accumulate result
    int out_index = (k + c * b);
    for (int i = 0; i < w * h; ++i) {
        int in_index = i + w * h * (k + b * c);
        sum += input[in_index]; // Accumulate the input values
    }
    output[out_index] = sum / (w * h); // Calculate the average
}