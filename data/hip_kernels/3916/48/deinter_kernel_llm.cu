#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate global index
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int totalElements = (NX + NY) * B;
    
    // Ensure the index is within bounds
    if (i >= totalElements) return;

    int b = i / (NX + NY); // Batch index
    int j = i % (NX + NY); // Element index within batch

    if (j < NX) {
        if (X) atomicAdd(&X[b * NX + j], OUT[i]); // Use atomic add to avoid race conditions
    } else {
        if (Y) atomicAdd(&Y[b * NY + j - NX], OUT[i]); // Use atomic add to avoid race conditions
    }
}
```
