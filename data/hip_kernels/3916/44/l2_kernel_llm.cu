#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // 1D grid, flattened for simpler calculation
    if(i < n){
        float diff = truth[i] - pred[i];
        error[i] = diff * diff; // Error calculated as squared difference
        delta[i] = diff; // Difference stored in delta
    }
}