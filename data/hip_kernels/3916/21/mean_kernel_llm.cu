#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const float scale = 1.f / (batch * spatial);
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread index is within the range
    if (i < filters) {
        float sum = 0.0f;
        
        // Use a single loop to replace nested loops for better coalescing
        for (int j = 0; j < batch * spatial; ++j) {
            sum += x[j * filters + i];
        }
        
        mean[i] = sum * scale;
    }
}