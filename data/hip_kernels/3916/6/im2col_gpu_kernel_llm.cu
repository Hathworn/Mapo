#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void im2col_gpu_kernel(const int n, const float* data_im, const int height, const int width, 
                                  const int ksize, const int pad, const int stride, 
                                  const int height_col, const int width_col, float *data_col) {
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimized loop using shared memory and avoiding bank conflicts
    extern __shared__ float shared_data[]; // Declare shared memory
    if (index < n) {
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        float* data_col_ptr = data_col + (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im + (channel_in * height + h_in) * width + w_in;

        for (int i = 0; i < ksize; ++i) {
            #pragma unroll // Unroll inner loop for performance
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                // Use shared memory for accessing image data
                shared_data[threadIdx.x] = (h >= 0 && w >= 0 && h < height && w < width) ? 
                                          data_im_ptr[i * width + j] : 0;
                __syncthreads(); // Ensure all threads have written

                // Write to global memory from shared memory
                *data_col_ptr = shared_data[threadIdx.x];
                data_col_ptr += height_col * width_col;
            }
        }
    }
}