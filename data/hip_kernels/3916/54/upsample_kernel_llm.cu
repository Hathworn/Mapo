#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x; // Compute unique thread index
    if(i >= N) return;
    
    // Decompose the index to multi-dimensional indices
    size_t tmp = i;
    int out_w = tmp % (w * stride); tmp /= (w * stride);
    int out_h = tmp % (h * stride); tmp /= (h * stride);
    int out_c = tmp % c;             tmp /= c;
    int b     = tmp % batch;

    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_c = out_c;

    int in_index = b * w * h * c + in_c * w * h + in_h * w + in_w;

    // Use conditional operation efficiently
    if(forward) {
        out[i] += scale * x[in_index]; // Use 'i' directly
    } else {
        atomicAdd(&x[in_index], scale * out[i]); // Use 'i' directly
    }
}