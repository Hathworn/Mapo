#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT) {
    // Calculate the global index of the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate additional offset based on 2D grid
    int offset = (blockIdx.y * gridDim.x) * blockDim.x;
    
    // Combine offsets to get true global index
    i += offset;

    // Avoid out-of-bounds memory access
    if (i < (NX + NY) * B) {
        int b = i / (NX + NY);  // Batch index
        int j = i % (NX + NY);  // Index within batch

        // Determine source array and index; fetch data accordingly
        OUT[i] = (j < NX) ? X[b * NX + j] : Y[b * NY + j - NX];
    }
}