#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    // Calculate the flat index and early return if out of bounds
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;
    
    // Use shared memory for intermediate sums within each warp
    extern __shared__ float shared_sum[];
    int b = index / spatial;
    int i = index % spatial;
    float sum = 0;

    // Compute sum of squares for each filter
    for (int f = 0; f < filters; ++f) {
        int idx = b * filters * spatial + f * spatial + i;
        sum += x[idx] * x[idx];
    }
    
    // Reduction within shared memory to calculate full sum
    shared_sum[threadIdx.x] = sum;
    __syncthreads();
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Normalize and calculate dx only by the first thread in the block
    if (threadIdx.x == 0) {
        sum = sqrtf(shared_sum[0]);
        if (sum == 0) sum = 1;
    }
    __syncthreads();
    sum = shared_sum[0];
    
    for (int f = 0; f < filters; ++f) {
        int idx = b * filters * spatial + f * spatial + i;
        x[idx] /= sum;
        dx[idx] = (1 - x[idx]) / sum;
    }
}