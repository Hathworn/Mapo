#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Calculate the global index using built-in functions
    int i = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
    
    // Ensure threads that exceed n do not proceed
    if (i < n) {
        binary[i] = (x[i] >= 0) ? 1 : -1;
    }
}