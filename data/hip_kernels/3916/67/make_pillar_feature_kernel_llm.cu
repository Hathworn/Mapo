#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void make_pillar_feature_kernel( float* dev_pillar_x_in_coors, float* dev_pillar_y_in_coors, float* dev_pillar_z_in_coors, float* dev_pillar_i_in_coors, float* dev_pillar_x, float* dev_pillar_y, float* dev_pillar_z, float* dev_pillar_i, int* dev_x_coors, int* dev_y_coors, float* dev_num_points_per_pillar, const int max_points, const int GRID_X_SIZE)
{
    int ith_pillar = blockIdx.x;
    int ith_point = threadIdx.x;
    if(ith_point >= dev_num_points_per_pillar[ith_pillar]) return; // Use early return for threads exceeding point count
    
    int x_ind = dev_x_coors[ith_pillar];
    int y_ind = dev_y_coors[ith_pillar];
    int coors_ind = y_ind * GRID_X_SIZE * max_points + x_ind * max_points + ith_point; // Precompute coors_ind index

    // Combine index calculations for dev_pillar_*
    int pillar_ind = ith_pillar * max_points + ith_point;
    dev_pillar_x[pillar_ind] = dev_pillar_x_in_coors[coors_ind];
    dev_pillar_y[pillar_ind] = dev_pillar_y_in_coors[coors_ind];
    dev_pillar_z[pillar_ind] = dev_pillar_z_in_coors[coors_ind];
    dev_pillar_i[pillar_ind] = dev_pillar_i_in_coors[coors_ind];
}