#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    float local_sum = 0.0f;  // Use a local variable to accumulate sum
    for(int k = 0; k < n; ++k){
        local_sum += x[k * groups + i];
    }
    sum[i] = local_sum;  // Write the result back to global memory only once
}