#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Optimize memory access with unrolling and boundary checks.
    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = id; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i;
            if (i < spatial) {
                local[id] += delta[index];
            }
        }
    }

    __syncthreads();

    // Accumulate reductions to a single thread.
    if(id == 0){
        mean_delta[filter] = 0;
        for(i = 0; i < threads; ++i){
            mean_delta[filter] += local[i];
        }
        mean_delta[filter] *= (-1.f/sqrtf(variance[filter] + .00001f));
    }
}