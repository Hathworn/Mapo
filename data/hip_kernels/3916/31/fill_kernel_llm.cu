#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplified 1D indexing
    int stride = gridDim.x * blockDim.x; // Calculate the stride for the loop
    for (int i = idx; i < N; i += stride) // Use loop unrolling for potential optimization
    {
        X[i*INCX] = ALPHA;
    }
}