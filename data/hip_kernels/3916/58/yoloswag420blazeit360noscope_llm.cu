#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Optimize block index calculation for 1D grid
    int stride = blockDim.x * gridDim.x;  // Calculate stride for next elements

    for (int i = id; i < size; i += stride) {  // Loop to process all elements
        input[i] = (rand[i] < prob) ? 0 : input[i] * scale;  // Conditional scaling
    }
}