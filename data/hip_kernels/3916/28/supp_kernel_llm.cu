#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize memory access pattern
    int stride = gridDim.x * blockDim.x;

    // Process elements in a strided loop
    for (int index = i; index < N; index += stride)
    {
        float value = X[index * INCX];
        if (value * value < ALPHA * ALPHA)
        {
            X[index * INCX] = 0;
        }
    }
}