#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1.f/(batch * spatial - 1);
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify global index calculation
    if (i >= filters) return;
    float sum = 0.0f;  // Use a local variable for accumulation
    for(int j = 0; j < batch; ++j) {
        for(int k = 0; k < spatial; ++k) {
            int index = j*filters*spatial + i*spatial + k;
            float diff = x[index] - mean[i];  // Avoid repeated calculation
            sum += diff * diff;  // Use multiplication instead of powf for performance
        }
    }
    variance[i] = sum * scale;  // Write back the result to the global memory
}