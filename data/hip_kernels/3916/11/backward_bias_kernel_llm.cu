#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Loop unrolling for efficiency
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; i += BLOCK*UNROLL_FACTOR) {
            #pragma unroll
            for (int u = 0; u < UNROLL_FACTOR; ++u) {
                int index = p + i + u * BLOCK + size * (filter + n * b);
                if (p + i + u * BLOCK < size) { // Check under condition
                    sum += delta[index];
                }
            }
        }
    }
    part[p] = sum;
    __syncthreads();

    // Parallel reduction to improve performance
    for (int stride = BLOCK / 2; stride > 0; stride /= 2) {
        if (p < stride) {
            part[p] += part[p + stride];
        }
        __syncthreads();
    }

    if (p == 0) {
        atomicAdd(&bias_updates[filter], part[0]); // Use atomicAdd for thread-safe accumulation
    }
}