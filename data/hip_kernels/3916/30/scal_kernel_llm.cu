#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop to handle multiple elements per thread if blockDim.x doesn't cover all elements
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x)
    {
        X[idx * INCX] *= ALPHA;
    }
}