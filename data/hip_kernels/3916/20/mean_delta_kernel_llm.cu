#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    // Calculate the index based on thread position
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float sum = 0.0f;
    int stride = filters * spatial;

    // Use stride access pattern to optimize global memory reads
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * stride + i * spatial + k;
            sum += delta[index];
        }
    }

    // Prevent redundant calculation in the loop
    mean_delta[i] = sum * (-1.f/sqrtf(variance[i] + .00001f));
}