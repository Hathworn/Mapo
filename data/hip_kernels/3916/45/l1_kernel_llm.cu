#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate the global index for the parallel thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within range
    if(i < n){
        // Compute difference between truth and prediction
        float diff = truth[i] - pred[i];
        
        // Update error using atomic operation to avoid race conditions
        error[i] = fabsf(diff);
        
        // Compute delta using ternary operator for simplicity
        delta[i] = (diff > 0) ? 1.0f : -1.0f;
    }
}