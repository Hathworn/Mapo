#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimized index calculation for 1D grid.
    if(i < n) {
        float t = truth[i];
        float p = pred[i];
        float log_p = logf(p + 1e-7f); // Use logf for better performance with floats.
        float log_one_minus_p = logf((1.0f - p) + 1e-7f); // Use logf for consistency and performance.
        error[i] = -t * log_p - (1.0f - t) * log_one_minus_p;
        delta[i] = t - p;
    }
}