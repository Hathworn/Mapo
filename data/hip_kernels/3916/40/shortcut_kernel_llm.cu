#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified ID calculation
    if (id >= size) return;

    // Calculate indices reversed for better performance
    int b = id / (minw * minh * minc);
    int remaining = id % (minw * minh * minc);
    int k = remaining / (minw * minh);
    remaining %= (minw * minh);
    int j = remaining / minw;
    int i = remaining % minw;

    // Reuse calculations and memory access
    int out_index_base = (j * sample + h2 * (k + c2 * b));
    int out_index = i * sample + w2 * out_index_base;
    int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));

    // Perform the operation
    out[out_index] = s1 * out[out_index] + s2 * add[add_index];
}