#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    if(i < n) {
        float dc_val = dc[i];
        float s_val = s[i];
        if(da) da[i] += dc_val * s_val;  // Update da if not null
        if(db) db[i] += dc_val * (1.0f - s_val);  // Update db if not null using 1.0f for float conversion
        ds[i] += dc_val * (a[i] - b[i]);  // Update ds
    }
}