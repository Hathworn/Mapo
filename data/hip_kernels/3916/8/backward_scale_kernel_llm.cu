#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    extern __shared__ float part[];  // Use dynamic shared memory allocation
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Unroll loop to enhance performance
    for(int b = 0; b < batch; ++b){
        for(int i = p; i < size; i += BLOCK){
            int index = i + size * (filter + n * b);
            sum += delta[index] * x_norm[index];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Use reduction for sum aggregation
    for(int stride = BLOCK / 2; stride > 0; stride /= 2) {
        if (p < stride) {
            part[p] += part[p + stride];
        }
        __syncthreads();
    }

    if (p == 0) {
        atomicAdd(&scale_updates[filter], part[0]);  // Atomic operation for better parallelism
    }
}