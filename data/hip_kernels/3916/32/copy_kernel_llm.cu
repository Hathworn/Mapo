#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate global thread ID more efficiently by using only blockIdx.x
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to minimize global memory access
    __shared__ float shared_X[256];  // Assuming blockDim.x <= 256

    if (i < N)
    {
        // Load data into shared memory
        shared_X[threadIdx.x] = X[i * INCX + OFFX];

        // Make sure all threads have loaded their data
        __syncthreads();

        // Store data from shared memory to global memory
        Y[i * INCY + OFFY] = shared_X[threadIdx.x];
    }
}