#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop through multiple elements if N is larger than the grid's capacity
    for(int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        Y[idx * INCY] *= X[idx * INCX]; // Perform multiplication
    }
}