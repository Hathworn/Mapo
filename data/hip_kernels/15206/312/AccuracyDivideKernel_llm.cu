#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory for efficiency
    __shared__ float sharedAccuracy;

    // Load accuracy into shared memory
    if (threadIdx.x == 0) {
        sharedAccuracy = *accuracy;
    }
    __syncthreads();

    // Divide in shared memory
    if (threadIdx.x == 0) {
        sharedAccuracy /= N;
    }
    __syncthreads();

    // Write back to global memory
    if (threadIdx.x == 0) {
        *accuracy = sharedAccuracy;
    }
}