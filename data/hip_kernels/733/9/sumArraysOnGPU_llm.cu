#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // Use shared memory to optimize memory access
    extern __shared__ float sharedMem[];

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        // Load data into shared memory
        sharedMem[threadIdx.x] = A[i];
        sharedMem[threadIdx.x + blockDim.x] = B[i];
        __syncthreads();

        // Perform the computation using shared memory
        C[i] = sharedMem[threadIdx.x] + sharedMem[threadIdx.x + blockDim.x];
    }
}