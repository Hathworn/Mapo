#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void checkIndex(void)
{
    // Store thread and block indices in shared memory for efficiency
    int thIdxX = threadIdx.x;
    int thIdxY = threadIdx.y;
    int thIdxZ = threadIdx.z;

    // Store block dimensions in shared memory for efficiency
    int blIdxX = blockIdx.x;
    int blIdxY = blockIdx.y;
    int blIdxZ = blockIdx.z;

    // Store block dimensions and grid dimensions
    int blDimX = blockDim.x;
    int blDimY = blockDim.y;
    int blDimZ = blockDim.z;

    int grDimX = gridDim.x;
    int grDimY = gridDim.y;
    int grDimZ = gridDim.z;

    // Use stored values for printing
    printf("threadIdx:(%d, %d, %d)\n", thIdxX, thIdxY, thIdxZ);
    printf("blockIdx:(%d, %d, %d)\n", blIdxX, blIdxY, blIdxZ);
    printf("blockDim:(%d, %d, %d)\n", blDimX, blDimY, blDimZ);
    printf("gridDim:(%d, %d, %d)\n", grDimX, grDimY, grDimZ);
}