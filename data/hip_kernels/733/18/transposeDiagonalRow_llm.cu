#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeDiagonalRow(float *out, float *in, const int nx, const int ny)
{
    // Calculate block index for diagonal using thread blocks
    unsigned int blk_x = blockIdx.x;
    unsigned int blk_y = blockIdx.y;

    // Compute global indices directly using block and thread indices
    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    // Perform transposition if within matrix boundaries
    if (ix < nx && iy < ny)
    {
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}