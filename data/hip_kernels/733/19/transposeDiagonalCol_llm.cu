#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeDiagonalCol(float *out, float *in, const int nx, const int ny)
{
    // Calculate global thread index using optimized approach
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure indices are within bounds before performing operations
    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}