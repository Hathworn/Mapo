#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll4Row(float *out, float *in, const int nx, const int ny)
{
    // Calculate indices with fewer operations
    unsigned int ix = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    // Use shared memory to reduce global memory accesses
    __shared__ float tile[4][4 + 1];  // Add padding to avoid bank conflicts

    if ((ix + 3 < nx) && (iy < ny))
    {
        // Each thread loads 4 elements into shared memory
        for (int i = 0; i < 4; i++)
        {
            tile[threadIdx.y][threadIdx.x + i * blockDim.x] = in[ti + i * blockDim.x];
        }
    }
    __syncthreads();

    // Only threads within bounds perform the write operation
    if ((iy + 3 * blockDim.y < ny) && (ix < nx))
    {
        for (int i = 0; i < 4; i++)
        {
            out[to + i * ny * blockDim.y] = tile[threadIdx.x + i * blockDim.x][threadIdx.y];
        }
    }
}