#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll4Col(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Precompute values to avoid redundant calculations
    unsigned int blockOffset = blockDim.x * ny;
    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        // Perform unrolled memory accesses efficiently
        out[ti]                = in[to];
        out[ti +   blockDim.x] = in[to +   blockOffset];
        out[ti + 2 * blockDim.x] = in[to + 2 * blockOffset];
        out[ti + 3 * blockDim.x] = in[to + 3 * blockOffset];
    }
}