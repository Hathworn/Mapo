#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Use a 2D grid and 2D blocks to better utilize GPU resources
    if (ix < nx && iy < ny)
    {
        int idx = iy * nx + ix;
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}