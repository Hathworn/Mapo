#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int stride = gridDim.x * blockDim.x; // Calculate stride for grid-stride loop

    // Use grid-stride loop for better memory access pattern
    for (int idx = ix; idx < nx * ny; idx += stride)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}