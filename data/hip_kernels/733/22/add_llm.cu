#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *c , int *d) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;  // Calculate global thread ID
    if (tid < N) { // Boundary check to ensure not accessing out of bounds
        d[tid] += c[tid];
    }
}