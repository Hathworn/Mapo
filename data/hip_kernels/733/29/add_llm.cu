#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add (float *d_A, float *d_B, float *d_C, int widthA, int widthB, int widthC)
{
    // Calculate single index for each dimension
    int idx = blockIdx.x * 64 + threadIdx.x * 2;
    int idy = blockIdx.y * 8 + threadIdx.y;

    // Compute starting locations for A, B, and C arrays
    int startA = idx + idy * widthA;
    int startB = idx + idy * widthB;
    int startC = idx + idy * widthC;

    // Load and sum float2 values
    float2 tempA = *(float2 *)(d_A + startA);
    float2 tempB = *(float2 *)(d_B + startB);
    tempA.x += tempB.x;
    tempA.y += tempB.y;

    // Store result into d_C
    *(float2 *)(d_C + startC) = tempA;
}