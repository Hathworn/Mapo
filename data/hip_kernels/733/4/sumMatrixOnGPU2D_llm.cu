#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Optimize calculation of unique index for better performance
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds before accessing memory
    if (ix < nx && iy < ny)
    {
        unsigned int idx = iy * nx + ix;
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}