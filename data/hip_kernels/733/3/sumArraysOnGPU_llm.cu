#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // Calculate global index 
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform operation if index is within bounds
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}