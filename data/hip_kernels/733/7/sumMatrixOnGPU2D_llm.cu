#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate unique thread index
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Simplified: bound check combined with quick return
    if (ix >= nx || iy >= ny) return;

    unsigned int idx = iy * nx + ix;

    // Perform the matrix addition
    MatC[idx] = MatA[idx] + MatB[idx];
}