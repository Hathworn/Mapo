#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyRow(float *out, float *in, const int nx, const int ny)
{
    // Calculate global thread index
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Use shared memory to temporarily store the row data
    extern __shared__ float sharedRow[];
    unsigned int threadIdxFlat = threadIdx.y * blockDim.x + threadIdx.x;
    
    if (ix < nx && iy < ny)
    {
        // Load data into shared memory
        sharedRow[threadIdxFlat] = in[iy * nx + ix];
        __syncthreads();
        
        // Copy from shared memory to global memory
        out[iy * nx + ix] = sharedRow[threadIdxFlat];
    }
}