#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeNaiveRow(float *out, float *in, const int nx, const int ny)
{
    // Use shared memory for faster access
    __shared__ float tile[32][33]; // 32x32 tile with padding to avoid bank conflicts

    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
    
    if (ix < nx && iy < ny)
    {
        // Load data into shared memory
        tile[threadIdx.y][threadIdx.x] = in[iy * nx + ix];
    }
    __syncthreads();

    // Transpose and store to global memory
    ix = blockDim.y * blockIdx.y + threadIdx.x;
    iy = blockDim.x * blockIdx.x + threadIdx.y;

    if (ix < ny && iy < nx)
    {
        out[ix * nx + iy] = tile[threadIdx.x][threadIdx.y];
    }
}