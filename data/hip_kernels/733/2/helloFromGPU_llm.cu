#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU()
{
    // Use a single thread to ensure only one printf execution
    if(threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Hello World from GPU!\n");
    }
}