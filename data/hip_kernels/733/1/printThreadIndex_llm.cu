#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void printThreadIndex(int *A, const int nx, const int ny)
{
    // Calculate thread's unique x and y coordinates in the grid
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Calculate the global index
    unsigned int idx = iy * nx + ix;

    // Ensure threads work within bounds
    if (ix < nx && iy < ny) {
        printf("thread_id (%d,%d) block_id (%d,%d) coordinate (%d,%d) global index"
        " %2d ival %2d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y,
        ix, iy, idx, A[idx]);
    }
}