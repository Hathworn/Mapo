#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNaiveCol(float *out, float *in, const int nx, const int ny)
{
    // Optimize block dimension calculations by using a shared variable
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check and transpose within valid index range
    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}