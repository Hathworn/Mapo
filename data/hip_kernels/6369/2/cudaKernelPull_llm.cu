#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaKernelPull(float* mat, float* spins, int size, float* temp, float tempStep, 
                               float* meanFieldElements, bool* continueIteration, float minDiff, 
                               int* unemptyCells, float linearCoef) {
    int blockId = blockIdx.x;
    int thrId = threadIdx.x;
    int globalId = blockId * size + thrId;

    // Optimize loop launch bounds and thread utilization
    extern __shared__ float sharedMeanFieldElements[];

    do {
        // Lessen temperature
        if (thrId == 0)
            temp[blockId] -= tempStep;

        // Stabilize
        do {
            // By default current iteration is the last one
            if (thrId == 0)
                continueIteration[blockId] = false;

            for (int spinId = 0; spinId < size; ++spinId) {
                __syncthreads();

                // Load to shared memory for better memory access pattern
                int wIndex = thrId;
                while (wIndex < unemptyCells[spinId * (size + 1)]) {
                    int idx = unemptyCells[spinId * (size + 1) + wIndex + 1];
                    sharedMeanFieldElements[wIndex] = spins[idx + blockId * size] * mat[spinId * size + idx];
                    wIndex += blockDim.x;
                }
                __syncthreads();

                // Parallel reduction using shared memory
                for (unsigned int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
                    if (thrId < offset && (thrId + offset) < unemptyCells[spinId * (size + 1)]) {
                        sharedMeanFieldElements[thrId] += sharedMeanFieldElements[thrId + offset];
                    }
                    __syncthreads();
                }
                
                if (thrId == 0) {
                    float meanField = sharedMeanFieldElements[0];
                    float old = spins[spinId + blockId * size];

                    if (temp[blockId] > 0) {
                        spins[spinId + blockId * size] = 
                            -1 * tanh(meanField / temp[blockId]) * linearCoef + 
                            spins[spinId + blockId * size] * (1 - linearCoef);
                    } else if (meanField > 0) {
                        spins[spinId + blockId * size] = -1;
                    } else {
                        spins[spinId + blockId * size] = 1;
                    }

                    // Check if we need another iteration
                    if (minDiff < fabs(old - spins[spinId + blockId * size])) {
                        continueIteration[blockId] = true;
                    }
                }
                __syncthreads();
            }
        } while (continueIteration[blockId]);
    } while (temp[blockId] >= 0);
}