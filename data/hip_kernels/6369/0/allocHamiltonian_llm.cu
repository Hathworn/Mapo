#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Global kernel function to allocate Hamiltonian
__global__ void allocHamiltonian(float* devMat, float* devSpins, int index, int size, double* energyTempor) {
    int wIndex = threadIdx.x + blockIdx.x * blockDim.x;

    // Optimize loop by precomputing index offsets and utilizing shared memory if beneficial
    while (wIndex < size * size) {
        int i = wIndex % size;
        int j = wIndex / size;  // Avoid explicit casting to int for performance
        double spinProduct = (double)(devSpins[i + index * size] * devSpins[j + index * size]);
        energyTempor[wIndex] = spinProduct * devMat[wIndex];
        wIndex += blockDim.x * gridDim.x;
    }
}