#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void quickSum(double* energyTempor, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int step = gridDim.x * blockDim.x;  // Calculate the total number of threads

    for (long long offset = 1; offset < size * size; offset *= 2) {
        // Use the calculated step size for indexing
        for (int i = index; (i * 2 + 1) * offset < size * size; i += step) {
            energyTempor[i * 2 * offset] += energyTempor[(i * 2 + 1) * offset];
        }
        __syncthreads();
    }
}