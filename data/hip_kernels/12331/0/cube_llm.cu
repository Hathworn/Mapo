#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Define the multithread action

// Start main activity
__global__ void cube(float *d_out, float *d_in) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    float f = d_in[idx];
    d_out[idx] = f * f * f;
}