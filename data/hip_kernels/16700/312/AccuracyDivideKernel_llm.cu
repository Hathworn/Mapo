#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Efficient global memory access
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    if (threadIdx.x == 0 && blockIdx.x == 0) { // Ensure only one thread performs the division
        *accuracy /= N;
    }
}