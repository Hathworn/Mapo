#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx(int ii, int d1, int d2, int d3, int d4, int scale_factor_t, int scale_factor_xy)
{
    int x, y, t, z, w;
    w = ii % d4;
    ii = ii / d4;
    z = ii % d3;
    ii = ii / d3;
    t = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w /= scale_factor_xy;  // Optimize division
    z /= scale_factor_xy;  // Optimize division
    t /= scale_factor_t;   // Optimize division
    d2 /= scale_factor_t;
    d3 /= scale_factor_xy;
    d4 /= scale_factor_xy;
    return (((((x * d1 + y) * d2) + t) * d3) + z) * d4 + w;
}

__global__ void upscale(float *input, float *output, long no_elements, int scale_factor_t, int scale_factor_xy, int d1, int d2, int d3, int d4)
{
    // Calculate unique global thread ID
    long ii = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y) * (blockDim.x * gridDim.x) + threadIdx.y * (blockDim.x * gridDim.x);
    if (ii >= no_elements) return;  // Exit if out of bounds

    int ipidx = translate_idx(ii, d1, d2, d3, d4, scale_factor_t, scale_factor_xy);  // Get input index
    output[ii] = input[ipidx];  // Transfer data
}