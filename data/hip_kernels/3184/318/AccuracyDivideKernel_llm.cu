#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load the value into a register for better performance
    float local_accuracy = *accuracy;
    
    // Perform division; use fast math if applicable
    local_accuracy /= N;
    
    // Write result back to global memory
    *accuracy = local_accuracy;
}