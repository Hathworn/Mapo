#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void minusTanh(float* out, const float* __restrict__ in1, const float* __restrict__ in2, int size){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure processing only valid elements and using memory coalescing
    if(id < size)
        out[id] = in1[id] - in2[id];
}