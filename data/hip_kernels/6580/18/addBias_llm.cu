#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addBias(float* Z, float* b, int Z_x_dim, int Z_y_dim) {
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Apply bias only for valid indices
    if (index < Z_x_dim * Z_y_dim) {
        int row = index / Z_x_dim;
        int col = index % Z_x_dim;
        Z[row * Z_x_dim + col] += b[row];
    }
}