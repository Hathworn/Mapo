#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SigmoidBackKernel(float* Z, float* dZ, int size) {
    // Calculate unique thread id
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure id is within the bounds
    if (id < size) {
        float t = Z[id];
        // Use fmaf for better precision and potential performance gain
        dZ[id] = dZ[id] * fmaf(-t, t, t);
    }
}