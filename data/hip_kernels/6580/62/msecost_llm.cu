#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void msecost(float* predictions, float* target, int size, float* cost) {
    __shared__ float shared_cost[256]; // Use shared memory for reduction
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Accumulate partial cost in shared memory
    float partial_cost = 0.0f;
    if (index < size) {
        float diff = predictions[index] - target[index];
        partial_cost = (diff * diff) / size;
    }
    shared_cost[tid] = partial_cost;
    __syncthreads();

    // Reduction within block to accumulate partial results
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_cost[tid] += shared_cost[tid + stride];
        }
        __syncthreads();
    }

    // Only one thread per block updates global cost
    if (tid == 0) {
        atomicAdd(cost, shared_cost[0]);
    }
}