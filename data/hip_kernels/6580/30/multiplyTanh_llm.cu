#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplyTanh(float* out, const float* __restrict__ in1, const float* __restrict__ in2, int size) {
    // Use shared memory for input data if beneficial (depending on further optimizations)
    extern __shared__ float sharedIn1[]; 
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop unrolling for better performance on large arrays
    for (int i = id; i < size; i += blockDim.x * gridDim.x) {
        out[i] = in1[i] * in2[i];
    }
}