#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void divideTanh(float* out, const float* __restrict__ in1, const float* __restrict__ in2, int size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a warp-level optimization to increase memory access efficiency
    if (id < size) {
        float val1 = __ldg(&in1[id]);
        float val2 = __ldg(&in2[id]);
        out[id] = val1 / val2;
    }
}