#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expMinus(float* out, float* in, int size) {
    // Use shared memory for input to improve memory access efficiency
    extern __shared__ float sharedIn[];

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Load input into shared memory
    if (id < size) {
        sharedIn[tid] = in[id];
    }
    __syncthreads();

    // Compute and store output
    if (id < size) {
        out[id] = __expf(-sharedIn[tid]);
    }
}