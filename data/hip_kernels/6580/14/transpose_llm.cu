#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_DIM 16

__global__ void transpose(float *odata, float *idata, int width, int height)
{
    __shared__ float block[BLOCK_DIM][BLOCK_DIM + 1];

    // Compute global indices for reading
    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;

    // Read from global memory and store into shared memory
    if((xIndex < width) && (yIndex < height))
    {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }

    // Synchronize to ensure all threads have written to shared memory
    __syncthreads();

    // Compute global indices for writing
    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;

    // Write transposed data from shared memory to global memory
    if((xIndex < height) && (yIndex < width))
    {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}