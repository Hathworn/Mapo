#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void g_getSoftMaxP(float* softMaxP, float* b, int cols, int row) {
    int bid = blockIdx.x;
    extern __shared__ float _share[];
    float* _max = _share;
    float* _sum = _share + blockDim.x;
    float* sp = softMaxP + bid * cols;

    // Initialize shared memory
    _sum[threadIdx.x] = 0.0f;
    _max[threadIdx.x] = -100000000.0f;

    // Find maximum value for the row
    for (int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
        int index = bid * cols + tid;
        sp[tid] = b[index];
        _max[threadIdx.x] = max(_max[threadIdx.x], sp[tid]);
    }
    __syncthreads();

    // Reduce max values
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            _max[threadIdx.x] = max(_max[threadIdx.x], _max[threadIdx.x + offset]);
        }
        __syncthreads();
    }

    // Subtract max, exponentiate, and compute sum
    for (int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
        sp[tid] = __expf(sp[tid] - _max[0]);
        _sum[threadIdx.x] += sp[tid];
    }
    __syncthreads();

    // Reduce sum values
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            _sum[threadIdx.x] += _sum[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Normalize the softmax probabilities
    for (int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
        sp[tid] /= _sum[0];
    }
}