#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reluActivationForward(float* Z, float* A, int Z_x_dim, int Z_y_dim) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;  // Calculate stride for loop unrolling
    for (int i = index; i < Z_x_dim * Z_y_dim; i += stride) {
        A[i] = fmaxf(Z[i], 0);  // Apply ReLU activation
    }
}