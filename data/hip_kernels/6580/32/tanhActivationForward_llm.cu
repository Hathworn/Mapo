#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tanhActivationForward(float* __restrict__ Z, float* __restrict__ A, int Z_x_dim, int Z_y_dim) {
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total_elements = Z_x_dim * Z_y_dim;
  
  // Use a loop to allow threads to handle multiple elements
  for (int i = index; i < total_elements; i += blockDim.x * gridDim.x) {
    A[i] = __builtin_tanhf(Z[i]); // Uses __builtin_tanhf for better performance
  }
}