#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateBiasKernel(float* dZ, float* b, int cols, int row, float learning_rate){
    int bid = blockIdx.x;
    extern __shared__ float _share[];
    float* _sum = _share;
    float* sp = dZ + cols * bid;
    _sum[threadIdx.x] = 0.0;
    
    // Use unrolling to speed up the reduction
    for(int id = threadIdx.x; id < cols; id += blockDim.x){
        _sum[threadIdx.x] += sp[id];
    }
    __syncthreads();

    // Optimized reduction using shared memory
    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if(threadIdx.x < offset) {
            _sum[threadIdx.x] += _sum[threadIdx.x + offset];
        }
        __syncthreads();
    }

    if(threadIdx.x == 0) {
        b[bid] -= learning_rate * (_sum[0] / cols);
    }
}