#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void oneMinusTanh(float* out, float* in, int size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check conditions to avoid redundant computations
    if (id < size) {
        float val = in[id];
        // Precompute 1 - val for efficiency
        out[id] = 1 - val;
    }
}