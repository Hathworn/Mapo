#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateBiasKernel_softmax(float* dZ, float* b, int cols, int row, float learning_rate) {
    int bid = blockIdx.x;
    extern __shared__ float _share[];
    // Use warp reduction for performance improvement
    float* sp = dZ + cols * bid;
    float local_sum = 0.0f;

    for (int id = threadIdx.x; id < cols; id += blockDim.x) {
        local_sum += sp[id];
    }

    _share[threadIdx.x] = local_sum;
    __syncthreads();

    // Perform reduction within the block
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            _share[threadIdx.x] += _share[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        b[bid] -= learning_rate * (_share[0] / cols);
    }
}