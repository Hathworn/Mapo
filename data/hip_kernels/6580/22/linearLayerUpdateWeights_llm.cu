#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void linearLayerUpdateWeights(float* dZ, float* A, float* W, int dZ_x_dim, int dZ_y_dim, int A_x_dim, int A_y_dim, float learning_rate) {

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int W_x_dim = A_y_dim;
    int W_y_dim = dZ_y_dim;

    if (row < W_y_dim && col < W_x_dim) {
        float dW_value = 0.0f;
        for (int i = 0; i < dZ_x_dim; i++) {
            // Optimize by reducing memory reads using registers
            float dZ_value = dZ[row * dZ_x_dim + i];
            float A_value = A[col * A_x_dim + i];
            dW_value += dZ_value * A_value;
        }
        // Perform update with learning rate outside the loop
        W[row * W_x_dim + col] -= learning_rate * (dW_value / A_x_dim);
    }
}