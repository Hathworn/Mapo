#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tanhActivationBackprop(float* Z, float* dA, float* dZ, int Z_x_dim, int Z_y_dim) {
    // Calculate the global index for the thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within the bounds of the input matrices
    if (index < Z_x_dim * Z_y_dim) {
        // Precompute reused value for performance
        float d = Z[index];
        float oneMinusSquareD = 1 - d * d;
        
        // Update dZ using the backpropagation formula
        dZ[index] = dA[index] * oneMinusSquareD;
    }
}