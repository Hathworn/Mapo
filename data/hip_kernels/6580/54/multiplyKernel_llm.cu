#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiplyKernel(float* Z, float* A, float* B, int size) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory for better memory access patterns
    __shared__ float sharedA[BLOCK_SIZE];
    __shared__ float sharedB[BLOCK_SIZE];

    if (id < size) {
        // Load data into shared memory
        sharedA[threadIdx.x] = A[id];
        sharedB[threadIdx.x] = B[id];
        __syncthreads();  // Ensure all data is loaded

        // Perform multiplication using shared memory
        Z[id] = sharedA[threadIdx.x] * sharedB[threadIdx.x];
    }
}