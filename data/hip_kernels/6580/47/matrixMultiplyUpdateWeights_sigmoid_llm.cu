#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__device__ float sigmoid(float x) {
    return 1.0f / (1 + __expf(-x));
}

__global__ void matrixMultiplyUpdateWeights_sigmoid(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns, float learning_rate) {
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y,
        tx = threadIdx.x, ty = threadIdx.y,
        Row = by * TILE_WIDTH + ty,
        Col = bx * TILE_WIDTH + tx;

    float Pvalue = 0;

    // Iterate through tiles of A and B
    for (int m = 0; m < (numAColumns + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
        // Collaborative loading of tiles into shared memory
        if (Row < numARows && m * TILE_WIDTH + tx < numAColumns) {
            ds_M[ty][tx] = A[Row * numAColumns + m * TILE_WIDTH + tx];
        } else {
            ds_M[ty][tx] = 0.0f;
        }
        
        if (Col < numBColumns && m * TILE_WIDTH + ty < numBRows) {
            ds_N[ty][tx] = B[(m * TILE_WIDTH + ty) * numBColumns + Col];
        } else {
            ds_N[ty][tx] = 0.0f;
        }

        __syncthreads();
        
        // Matrix multiplication within the tile
        for (int k = 0; k < TILE_WIDTH; ++k) {
            Pvalue += ds_M[ty][k] * ds_N[k][tx];
        }
        
        __syncthreads();
    }

    // Write the result to the output matrix
    if (Row < numCRows && Col < numCColumns) {
        C[Row * numCColumns + Col] -= learning_rate * (Pvalue / numAColumns);
    }
}