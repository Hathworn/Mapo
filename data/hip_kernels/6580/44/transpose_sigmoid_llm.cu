#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float sigmoid(float x) {
    return 1.0f / (1 + __expf(-x));
}

__global__ void transpose_sigmoid(float *odata, float *idata, int width, int height) {
    __shared__ float block[BLOCK_DIM][BLOCK_DIM+1];

    // Calculate matrix indices
    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;

    // Load one element per thread into shared memory
    if (xIndex < width && yIndex < height) {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = sigmoid(idata[index_in]); // Apply sigmoid during load
    }

    __syncthreads(); // Ensure all writes to block[][] have completed

    // Transpose write to global memory
    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;
    if (xIndex < height && yIndex < width) {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y]; // Transpose write
    }
}