#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TanhBackKernel(float* Z, float* dZ, int size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for potential performance gains (assuming size is small enough)
    __shared__ float sharedZ[1024]; // Adjust size according to blockDim.x
    __shared__ float sharedDZ[1024]; // Adjust size according to blockDim.x

    if (id < size) {
        // Load data into shared memory
        sharedZ[threadIdx.x] = Z[id];
        sharedDZ[threadIdx.x] = dZ[id];
        __syncthreads();

        // Perform computation using shared memory
        float t = sharedZ[threadIdx.x];
        dZ[id] = sharedDZ[threadIdx.x] * (1 - t * t);
    }
}