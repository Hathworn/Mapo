#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_optimized(float* softmaxP, float* b, int rows, int cols){
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    __shared__ float shared_max;
    __shared__ float shared_sum;

    if(tid < rows){
        // Initialize shared variables for reduction operations
        shared_max = -100000000.0f;
        shared_sum = 0.0f;
        __syncthreads();

        // Calculate max value using a block-wide reduction
        atomicMax(&shared_max, b[tid * cols + bid]);
        __syncthreads();

        // Compute softmaxP using the max value
        float val = b[tid * cols + bid] - shared_max;
        val = __expf(val);

        // Accumulate the sum in shared memory
        atomicAdd(&shared_sum, val);
        __syncthreads();

        // Normalize the probabilities
        if (shared_sum > 0.0f) {
            softmaxP[tid * cols + bid] = val / shared_sum;
        }
    }
}