#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_linear(float* softmaxP, float* b, int rows, int cols) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int gid = tid + blockDim.x * bid;

    extern __shared__ float _share[];

    if (gid < rows * cols) {
        // Load data into shared memory
        float val = b[gid];
        _share[tid] = val;
        __syncthreads();

        // Compute max
        float _max = val;
        for (int offset = 1; offset < blockDim.x; offset <<= 1) {
            if (tid + offset < rows) {
                _max = max(_max, _share[tid + offset]);
            }
        }
        __syncthreads();

        // Compute exponentials and sum
        float exp_val = __expf(_share[tid] - _max);
        _share[tid] = exp_val;
        sum = exp_val;
        
        for (int offset = 1; offset < blockDim.x; offset <<= 1) {
            if (tid + offset < rows) {
                sum += _share[tid + offset];
            }
        }
        __syncthreads();

        // Write results to output
        softmaxP[gid] = _share[tid] / sum;
    }
}