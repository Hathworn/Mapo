#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReluBackKernel(float* Z, float* dZ, int size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Check within grid-stride loop for better efficiency
    for (int idx = id; idx < size; idx += blockDim.x * gridDim.x) {
        // Use a single statement for condition
        dZ[idx] = Z[idx] > 0 ? dZ[idx] : 0;
    }
}