#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_DIM 16  // assuming BLOCK_DIM is defined elsewhere

__global__ void transpose_softmax(float *odata, float *idata, int width, int height)
{
    __shared__ float block[BLOCK_DIM][BLOCK_DIM + 1];

    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;

    // Utilize shared memory efficiently by avoiding bank conflicts
    if (xIndex < width && yIndex < height)
    {
        block[threadIdx.y][threadIdx.x] = idata[yIndex * width + xIndex];
    }

    __syncthreads();

    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;

    // Ensure that the transposed indices access within bounds
    if (xIndex < height && yIndex < width)
    {
        odata[yIndex * height + xIndex] = block[threadIdx.x][threadIdx.y];
    }
}