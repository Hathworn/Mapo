#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiplyAndTanh(float * A, float * B, float * C, float* b, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
    
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
    float Pvalue = 0;

    // Loop over TILE_WIDTH-sized sub-matrices
    for (int m = 0; m < (numAColumns + TILE_WIDTH - 1)/TILE_WIDTH; ++m) {
        // Load A into shared memory
        if (Row < numARows && m*TILE_WIDTH + tx < numAColumns) {
            ds_M[ty][tx] = A[Row * numAColumns + m*TILE_WIDTH + tx];
        } else {
            ds_M[ty][tx] = 0.0f;
        }
        
        // Load B into shared memory
        if (Col < numBColumns && m*TILE_WIDTH + ty < numBRows) {
            ds_N[ty][tx] = B[(m*TILE_WIDTH + ty) * numBColumns + Col];
        } else {
            ds_N[ty][tx] = 0.0f;
        }

        __syncthreads();

        // Matrix multiplication
        for (int k = 0; k < TILE_WIDTH; ++k) {
            Pvalue += ds_M[ty][k] * ds_N[k][tx];
        }
        
        __syncthreads();
    }

    // Apply tanh activation function
    if (Row < numCRows && Col < numCColumns) {
        C[Row*numCColumns + Col] = tanhf(Pvalue + b[Row]); // Use fast math function
    }
}