#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expPlus(float* out, float* in, int size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Coalesced memory access pattern by using loop unrolling
    for (; id < size; id += blockDim.x * gridDim.x) {
        out[id] = __expf(in[id]);
    }
}