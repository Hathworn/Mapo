#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float sigmoid(float x) {
    return 1.0f / (1 + __expf(-x));
}

__global__ void updateBiasKernel_sigmoid(float* dZ, float* b, int cols, int row, float learning_rate) {
    extern __shared__ float _sum[];
    int bid = blockIdx.x;
    float* sp = dZ + cols * bid;
    
    // Initialize shared memory
    _sum[threadIdx.x] = 0.0f;

    // Sum data for each thread
    for (int id = threadIdx.x; id < cols; id += blockDim.x) {
        _sum[threadIdx.x] += sp[id];
    }
    __syncthreads();

    // Reduction using parallel summation
    for (int len = blockDim.x >> 1; len > 0; len >>= 1) {
        if (threadIdx.x < len) {
            _sum[threadIdx.x] += _sum[threadIdx.x + len];
        }
        __syncthreads();
    }

    // Update bias for the first thread in the block
    if (threadIdx.x == 0) {
        b[bid] -= learning_rate * (_sum[0] / cols);
    }
}