#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void divideKernel(float* A, int size) {
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread acts only on valid data range, using warp divergence minimization
    if (id < size) {
        A[id] = __fdividef(1.0f, A[id]);  // Use fast float division intrinsic
    }
}