#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initializeBiasKernel_softmax(float* b, int size) {
    // Use a more efficient stride loop for larger datasets
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
        b[index] = 0.0f;
    }
}