#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void linearLayerUpdateBias(float* dZ, float* b, int dZ_x_dim, int dZ_y_dim, int b_x_dim, float learning_rate) {
    int dZ_y = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Iterate over the x dimension within a thread
    if (dZ_y < dZ_y_dim) {
        float sum = 0.0f;
        for (int dZ_x = 0; dZ_x < dZ_x_dim; ++dZ_x) {
            sum += dZ[dZ_y * dZ_x_dim + dZ_x];
        }
        // Use atomic operation to update b
        atomicAdd(&b[dZ_y], -learning_rate * (sum / dZ_x_dim));
    }
}