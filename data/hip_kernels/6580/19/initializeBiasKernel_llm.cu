#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initializeBiasKernel(float* b, int size) {
    // Use shared memory for faster memory access
    extern __shared__ float sharedBias[];

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        sharedBias[threadIdx.x] = 0.0;
        __syncthreads();
        b[index] = sharedBias[threadIdx.x];
    }
}