#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void plusTanh(float* out, float* in1, float* in2, int size){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds before proceeding with computation
    if(id < size) {
        float result = in1[id] + in2[id];
        out[id] = tanh(result); // Apply tanh directly to the result
    }
}