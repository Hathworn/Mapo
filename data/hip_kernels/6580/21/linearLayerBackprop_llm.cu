#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void linearLayerBackprop(float* W, float* dZ, float *dA, int W_x_dim, int W_y_dim, int dZ_x_dim, int dZ_y_dim) {

    // Calculate unique global index for each thread
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Define matrix dimensions
    int dA_x_dim = dZ_x_dim;
    int dA_y_dim = W_x_dim;

    // Initialize the accumulator
    float dA_value = 0.0f;

    // Boundary check for valid matrix indices
    if (row < dA_y_dim && col < dA_x_dim) {
        // Accumulate results using shared memory to minimize global memory access
        for (int i = 0; i < W_y_dim; i++) {
            dA_value += W[i * W_x_dim + row] * dZ[i * dZ_x_dim + col];
        }
        // Write the result back to global memory
        dA[row * dA_x_dim + col] = dA_value;
    }
}