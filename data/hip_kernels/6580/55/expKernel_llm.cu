#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expKernel(float* Z, float* A, int size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid-stride loop to ensure efficient processing of larger data sizes
    for (int i = id; i < size; i += blockDim.x * gridDim.x) {
        A[i] = __expf(-Z[i]);
    }
}