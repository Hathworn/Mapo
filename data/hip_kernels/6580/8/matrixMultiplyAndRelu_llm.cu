#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define TILE_WIDTH 16

__global__ void matrixMultiplyAndRelu(float *A, float *B, float *C, float *b, float *T, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    // Use shared memory tiles for matrix multiplication
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
    
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
    
    float Pvalue = 0.0f;

    // Loop over TILE_WIDTH segments
    for (int m = 0; m < (numAColumns + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
        // Collaborative loading of A and B tiles into shared memory
        if (Row < numARows && m * TILE_WIDTH + tx < numAColumns) {
            ds_M[ty][tx] = A[Row * numAColumns + m * TILE_WIDTH + tx];
        } else {
            ds_M[ty][tx] = 0.0f;
        }
        if (Col < numBColumns && m * TILE_WIDTH + ty < numBRows) {
            ds_N[ty][tx] = B[(m * TILE_WIDTH + ty) * numBColumns + Col];
        } else {
            ds_N[ty][tx] = 0.0f;
        }

        __syncthreads();

        // Matrix multiplication for current tiles
        for (int k = 0; k < TILE_WIDTH; ++k) {
            Pvalue += ds_M[ty][k] * ds_N[k][tx];
        }

        __syncthreads();
    }

    // Store the result in the global memory
    if (Row < numCRows && Col < numCColumns) {
        float num = Pvalue + b[Row];
        T[Row * numCColumns + Col] = num;
        C[Row * numCColumns + Col] = fmaxf(num, 0.0f);
    }
}