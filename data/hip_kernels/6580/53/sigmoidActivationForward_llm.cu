#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float sigmoid(float x) {
    return 1.0f / (1 + __expf(-x));
}

__global__ void sigmoidActivationForward(float* Z, float* A, int Z_x_dim, int Z_y_dim) {

    // Calculate the global index and ensure all threads are utilized
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop through the elements in steps of total threads
    for (int i = index; i < Z_x_dim * Z_y_dim; i += stride) {
        A[i] = sigmoid(Z[i]);
    }
}