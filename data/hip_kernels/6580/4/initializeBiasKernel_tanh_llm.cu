#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initializeBiasKernel_tanh(float* b, int size) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use stride loop to ensure all elements are initialized
    while (index < size) {
        b[index] = 0.0f; // Use 0.0f for better floating-point consistency
        index += stride;
    }
}