#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__global__ void matrixMultiply(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float Pvalue = 0;

    for (int m = 0; m < (numAColumns - 1) / TILE_WIDTH + 1; ++m) {
        // Load A and B tiles into shared memory
        int tiledRow = m * TILE_WIDTH + tx;
        int tiledCol = m * TILE_WIDTH + ty;
        
        ds_M[ty][tx] = (Row < numARows && tiledRow < numAColumns) ? A[Row * numAColumns + tiledRow] : 0;
        ds_N[ty][tx] = (Col < numBColumns && tiledCol < numBRows) ? B[tiledCol * numBColumns + Col] : 0;

        __syncthreads();

        // Compute partial product
        for (int k = 0; k < TILE_WIDTH; ++k) {
            Pvalue += ds_M[ty][k] * ds_N[k][tx];
        }

        __syncthreads();
    }
    // Write the result to C
    if (Row < numCRows && Col < numCColumns) {
        C[Row * numCColumns + Col] = Pvalue;
    }
}