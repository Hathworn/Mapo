#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiplyUpdateWeights(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns, float learning_rate) {
    // Shared memory to enhance data reuse
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

    // Block and thread indices
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
    float Pvalue = 0;

    // Loop over the tiles of A and B in phases
    for (int m = 0; m < (numAColumns + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
        // Load tile of matrix A into shared memory
        if (Row < numARows && m * TILE_WIDTH + tx < numAColumns)
            ds_M[ty][tx] = A[Row * numAColumns + m * TILE_WIDTH + tx];
        else
            ds_M[ty][tx] = 0.0;

        // Load tile of matrix B into shared memory
        if (Col < numBColumns && m * TILE_WIDTH + ty < numBRows)
            ds_N[ty][tx] = B[(m * TILE_WIDTH + ty) * numBColumns + Col];
        else
            ds_N[ty][tx] = 0.0;

        // Synchronize to ensure all tiles are loaded
        __syncthreads();
        
        // Compute contribution of this tile to P
        for (int k = 0; k < TILE_WIDTH; ++k)
            Pvalue += ds_M[ty][k] * ds_N[k][tx];

        // Synchronize to wait for computation before loading next tile
        __syncthreads();
    }

    // Write the result into matrix C
    if (Row < numCRows && Col < numCColumns)
        C[Row * numCColumns + Col] -= learning_rate * (Pvalue / numAColumns);
}