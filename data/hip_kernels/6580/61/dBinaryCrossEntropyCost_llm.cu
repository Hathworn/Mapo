#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dBinaryCrossEntropyCost(float* predictions, float* target, float* dY, int size) {
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Bounds check
    if (index >= size) return;
    
    // Efficient memory access and storing result
    float pred = predictions[index];
    float targ = target[index];
    dY[index] = -1.0f * (targ / pred - (1 - targ) / (1 - pred));
}