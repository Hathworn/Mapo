#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void linearLayerForward(float* W, float* A, float* Z, float* b, int W_x_dim, int W_y_dim, int A_x_dim, int A_y_dim) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int Z_x_dim = A_x_dim;
    int Z_y_dim = W_y_dim;

    float Z_value = 0;

    // Use shared memory to optimize memory access
    __shared__ float shared_W[32][32]; // Adjust size as needed
    __shared__ float shared_A[32][32]; // Adjust size as needed

    for (int i = 0; i < (W_x_dim + 31) / 32; ++i) { // Loop over tiles
        if (i * 32 + threadIdx.x < W_x_dim && row < Z_y_dim) {
            shared_W[threadIdx.y][threadIdx.x] = W[row * W_x_dim + i * 32 + threadIdx.x];
        } else {
            shared_W[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (i * 32 + threadIdx.y < W_x_dim && col < Z_x_dim) {
            shared_A[threadIdx.y][threadIdx.x] = A[(i * 32 + threadIdx.y) * A_x_dim + col];
        } else {
            shared_A[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for (int j = 0; j < 32; ++j) {
            Z_value += shared_W[threadIdx.y][j] * shared_A[j][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < Z_y_dim && col < Z_x_dim) {
        Z[row * Z_x_dim + col] = Z_value + b[row];
    }
}