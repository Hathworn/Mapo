#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateBiasKernel_tanh(float* dZ, float* b, int cols, int row, float learning_rate) {
    int bid = blockIdx.x;
    extern __shared__ float _share[];
    float* _sum = _share;
    float* sp = dZ + cols * bid;
    _sum[threadIdx.x] = 0.0;

    // Parallel reduction using shared memory
    for (int id = threadIdx.x; id < cols; id += blockDim.x) {
        _sum[threadIdx.x] += sp[id];
    }
    __syncthreads();

    // Reduce shared memory values
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            _sum[threadIdx.x] += _sum[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Update bias using the result from the reduction
    if (threadIdx.x == 0) {
        b[bid] -= learning_rate * (_sum[0] / cols);
    }
}