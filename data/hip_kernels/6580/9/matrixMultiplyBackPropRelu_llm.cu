#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__global__ void matrixMultiplyBackPropRelu(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    // Shared memory for storing sub-matrices
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
    
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
    
    float Pvalue = 0;

    // Loop over all tiles required to compute C element
    for (int m = 0; m < (numAColumns + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
        // Load data into shared memory; zero-padding boundaries
        if (Row < numARows && m * TILE_WIDTH + tx < numAColumns)
            ds_M[ty][tx] = A[Row * numAColumns + m * TILE_WIDTH + tx];
        else
            ds_M[ty][tx] = 0;

        if (Col < numBColumns && m * TILE_WIDTH + ty < numBRows)
            ds_N[ty][tx] = B[(m * TILE_WIDTH + ty) * numBColumns + Col];
        else
            ds_N[ty][tx] = 0;
        
        __syncthreads();

        // Perform computation within the tile
        for (int k = 0; k < TILE_WIDTH; ++k)
            Pvalue += ds_M[ty][k] * ds_N[k][tx];
        
        __syncthreads();
    }
    
    // Write result to the output matrix
    if (Row < numCRows && Col < numCColumns)
        C[Row * numCColumns + Col] = Pvalue;
}