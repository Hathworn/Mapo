#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateBiasKernel_relu(float* dZ, float* b, int cols, int row, float learning_rate) {
    int bid = blockIdx.x;
    extern __shared__ float _share[];
    float * _sum = _share;
    float* sp = dZ + cols * bid;
    _sum[threadIdx.x] = 0.0;

    for (int id = threadIdx.x; id < cols; id += blockDim.x) {
        // Utilize warp-level primitives to efficiently reduce the sum
        _sum[threadIdx.x] += sp[id];
    }
    __syncthreads();
    
    // Parallel reduction using warp shuffle
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        _sum[threadIdx.x] += __shfl_down_sync(0xFFFFFFFF, _sum[threadIdx.x], offset);
    }

    // Store final result by leader thread of the warp
    if (threadIdx.x % warpSize == 0) {
        atomicAdd(&b[bid], -learning_rate * (_sum[threadIdx.x] / cols));
    }
}