#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reluActivationBackprop(float* Z, float* dA, float* dZ, int Z_x_dim, int Z_y_dim) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_size = Z_x_dim * Z_y_dim;

    // Use threadIdx and blockIdx for parallel execution and boundary checking
    if (index < total_size) {
        // Use ternary operator for concise conditional operation
        dZ[index] = Z[index] > 0 ? dA[index] : 0;
    }
}