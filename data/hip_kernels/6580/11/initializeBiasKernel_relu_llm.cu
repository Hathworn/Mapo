#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initializeBiasKernel_relu(float* b, int size) {
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a stride loop for better coalescing and efficiency
    for (int i = index; i < size; i += blockDim.x * gridDim.x) {
        b[i] = 0.0;
    }
}