#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dMSECost(float* predictions, float* target, float* dY, int size) {
    // Calculate global index with inline calculation
    int index = (blockIdx.x * blockDim.x + threadIdx.x);

    // Use the warp size for loading efficiency
    if (index < size) {
        // Optimize computation - directly multiply and store result
        dY[index] = __fmaf_rn(2.0f, (predictions[index] - target[index]), 0.0f);
    }
}