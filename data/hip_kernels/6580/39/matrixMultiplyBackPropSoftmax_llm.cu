#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__global__ void matrixMultiplyBackPropSoftmax(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    // Shared memory for tiles of matrices A and B
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
    
    float Pvalue = 0;

    // Loop over tiles of A and B
    for (int m = 0; m < (numAColumns + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
        // Load tiles into shared memory
        if (Row < numARows && m*TILE_WIDTH + tx < numAColumns)
            ds_M[ty][tx] = A[Row * numAColumns + m * TILE_WIDTH + tx];
        else
            ds_M[ty][tx] = 0.0f;
        
        if (Col < numBColumns && m*TILE_WIDTH + ty < numBRows)
            ds_N[ty][tx] = B[(m * TILE_WIDTH + ty) * numBColumns + Col];
        else
            ds_N[ty][tx] = 0.0f;

        __syncthreads();

        // Compute partial result
        for (int k = 0; k < TILE_WIDTH; ++k) {
            Pvalue += ds_M[ty][k] * ds_N[k][tx];
        }

        __syncthreads();
    }

    // Write the result
    if (Row < numCRows && Col < numCColumns) 
        C[Row * numCColumns + Col] = Pvalue;
}