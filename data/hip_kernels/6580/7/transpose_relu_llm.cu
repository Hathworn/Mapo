#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCK_DIM 16

__global__ void transpose_relu(float *odata, float *idata, int width, int height)
{
    __shared__ float block[BLOCK_DIM][BLOCK_DIM + 1]; // Optimize alignment for shared memory

    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;

    // Reduce redundant condition checks
    if (xIndex < width && yIndex < height) 
    {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in]; // Load data into shared memory
    }

    __syncthreads(); // Ensure all writes to block[][] have completed

    // Use more descriptive variable names
    unsigned int xIndex_out = blockIdx.y * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex_out = blockIdx.x * BLOCK_DIM + threadIdx.y;
    if (xIndex_out < height && yIndex_out < width) 
    {
        unsigned int index_out = yIndex_out * height + xIndex_out;
        odata[index_out] = block[threadIdx.x][threadIdx.y]; // Write transposed data to global memory
    }
}