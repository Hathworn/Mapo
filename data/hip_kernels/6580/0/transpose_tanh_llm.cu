#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_DIM 16

__global__ void transpose_tanh(float *odata, float *idata, int width, int height)
{
    __shared__ float block[BLOCK_DIM][BLOCK_DIM+1];

    // Calculate global indices
    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;

    // Load from idata to shared memory ensuring boundary conditions
    if (xIndex < width && yIndex < height)
    {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }

    __syncthreads(); // Ensure all threads have written to shared memory

    // Calculate transposed global indices
    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;

    // Store from shared memory to odata with boundary check
    if (xIndex < height && yIndex < width)
    {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}