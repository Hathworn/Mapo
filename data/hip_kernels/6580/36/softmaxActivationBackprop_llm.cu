#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmaxActivationBackprop(float* Z, float* dA, float* dZ, int Z_x_dim, int Z_y_dim) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Ensure all elements are processed with stride-based looping
    for (int i = index; i < Z_x_dim * Z_y_dim; i += stride) {
        dZ[i] = dA[i];
    }
}