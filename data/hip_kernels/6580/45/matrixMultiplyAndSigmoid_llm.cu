#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__device__ float sigmoid(float x) {
    return 1.0f / (1 + __expf(-x));
}

__global__ void matrixMultiplyAndSigmoid(float* A, float* B, float* C, float* b, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float Pvalue = 0;

    // Loop over tiles
    for (int m = 0; m < (numAColumns + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
        if (Row < numARows && m * TILE_WIDTH + tx < numAColumns)
            ds_M[ty][tx] = A[Row * numAColumns + m * TILE_WIDTH + tx];
        else
            ds_M[ty][tx] = 0;

        if (Col < numBColumns && m * TILE_WIDTH + ty < numBRows)
            ds_N[ty][tx] = B[(m * TILE_WIDTH + ty) * numBColumns + Col];
        else
            ds_N[ty][tx] = 0;

        __syncthreads();

        // Unroll the loop for better performance
        for (int k = 0; k < TILE_WIDTH; k += 4) {
            Pvalue += ds_M[ty][k] * ds_N[k][tx];
            Pvalue += ds_M[ty][k + 1] * ds_N[k + 1][tx];
            Pvalue += ds_M[ty][k + 2] * ds_N[k + 2][tx];
            Pvalue += ds_M[ty][k + 3] * ds_N[k + 3][tx];
        }

        __syncthreads();
    }

    // Write result to matrix C
    if (Row < numCRows && Col < numCColumns)
        C[Row * numCColumns + Col] = sigmoid(Pvalue + b[Row]);
}
```
