#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void minusKernel(float* A, int size) {
    // Use grid-stride loop for better utilization
    for (int id = blockIdx.x * blockDim.x + threadIdx.x; id < size; id += blockDim.x * gridDim.x) {
        A[id] = 1 - A[id];
    }
}