#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__global__ void matrixMultiplyBackPropSigmoid(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    // Improved shared memory usage
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];
    
    int bx = blockIdx.x, by = blockIdx.y,
        tx = threadIdx.x, ty = threadIdx.y,
        Row = by * TILE_WIDTH + ty,
        Col = bx * TILE_WIDTH + tx;
    
    float Pvalue = 0.0f;

    // Reduce the number of iterations by precalculating iteration count
    int numTiles = (numAColumns + TILE_WIDTH - 1) / TILE_WIDTH;

    for (int m = 0; m < numTiles; ++m) {
        // Coalesced memory access
        if (Row < numARows && m * TILE_WIDTH + tx < numAColumns) {
            ds_M[ty][tx] = A[Row * numAColumns + m * TILE_WIDTH + tx];
        } else {
            ds_M[ty][tx] = 0.0f;
        }

        if (Col < numBColumns && m * TILE_WIDTH + ty < numBRows) {
            ds_N[ty][tx] = B[(m * TILE_WIDTH + ty) * numBColumns + Col];
        } else {
            ds_N[ty][tx] = 0.0f;
        }

        __syncthreads();

        // Loop unrolling and optimal register usage
        for (int k = 0; k < TILE_WIDTH; k++) {
            Pvalue += ds_M[ty][k] * ds_N[k][tx];
        }

        __syncthreads();
    }

    // More optimal boundary check
    if (Row < numCRows && Col < numCColumns) {
        C[Row * numCColumns + Col] = Pvalue;
    }
}