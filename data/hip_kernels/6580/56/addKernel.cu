#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(float* A, int size){
int id = blockIdx.x * blockDim.x + threadIdx.x;

if(id < size){
A[id] = 1 + A[id];
}
}