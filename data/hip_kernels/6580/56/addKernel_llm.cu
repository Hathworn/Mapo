#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(float* A, int size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if the ID is within bounds
    if (id >= size) return;

    // Perform addition
    A[id] += 1;
}