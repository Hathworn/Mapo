#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float sigmoid(float x) {
    return 1.0f / (1 + __expf(-x));
}

__global__ void sigmoidActivationBackprop(float* Z, float* dA, float* dZ, int Z_x_dim, int Z_y_dim) {
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = Z_x_dim * Z_y_dim;

    // Loop to cover entire matrix using grid stride
    for (; index < total_elements; index += blockDim.x * gridDim.x) {
        float sigmoid_value = sigmoid(Z[index]);
        dZ[index] = dA[index] * sigmoid_value * (1 - sigmoid_value);
    }
}