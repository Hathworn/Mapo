#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float sigmoid(float x) {
    return 1.0f / (1 + __expf(-x));
}

__global__ void initializeBiasKernel_sigmoid(float* b, int size) {
    // Use grid-stride loop for better utilization of GPU resources
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    for (int i = index; i < size; i += stride) {
        b[i] = 0.0;
    }
}