#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binaryCrossEntropyCost(float* predictions, float* target, int size, float* cost) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread computes its own contribution to the cost
    if (index < size) {
        float pred = predictions[index];
        float tgt = target[index];
        
        // Reduce number of computations by storing shared operations
        float log_pred = logf(pred);
        float log_one_minus_pred = logf(1.0f - pred);
        
        float partial_cost = tgt * log_pred + (1.0f - tgt) * log_one_minus_pred;
        
        // Use atomic operations to sum the contributions of all threads
        atomicAdd(cost, - partial_cost / size);
    }
}