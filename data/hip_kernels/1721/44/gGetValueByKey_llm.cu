#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gGetValueByKey(float* d_in, float* d_out, int* indeces, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread is within bounds
    if (tid >= n) return;

    // Read index once to minimize global memory accesses
    int index = indeces[tid];

    // Perform the assignment
    d_out[tid] = d_in[index];
}