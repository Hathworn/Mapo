#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gScatterAdd(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure idx is within bounds of sparseSize
    if (idx < sparseSize) {
        int sparseIndex = sparseIndices[idx] + offset;

        // Check if sparseIndex is within dense bounds
        if (sparseIndex >= 0 && sparseIndex < denseSize) {
            // Atomic operation for thread-safe addition
            atomicAdd(&denseData[sparseIndex], sparseData[idx]);
        }
    }
}