#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel by reducing divergence and accessing memory coalesced
__global__ void gGather(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // Use shared memory for faster data access
    __shared__ int sharedSparseIndices[1024]; // Adjust size as needed based on block size
    
    if (idx < sparseSize) {
        sharedSparseIndices[threadIdx.x] = sparseIndices[idx];
        __syncthreads(); // Ensure all threads have loaded their values
        
        int sparseIndex = sharedSparseIndices[threadIdx.x];
        if (sparseIndex >= -offset && sparseIndex + offset < denseSize) {
            sparseData[idx] = denseData[sparseIndex + offset];
        }
    }
}