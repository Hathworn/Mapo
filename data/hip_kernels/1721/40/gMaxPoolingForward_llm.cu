#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gMaxPoolingForward(float* out, int outRows, int outCols, float* in, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Optimize: Check valid thread index
    if(tid >= outRows * outCols)
        return;

    int rowId = tid / outCols; // Corrected to use outCols for rowId calculation
    int colId = tid % outCols; // Corrected to use outCols for colId calculation

    // Adjust pointers for input and mask
    float* b = in + (rowId * inCols) + (colId * width);
    float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;

    // Use "width" directly within the loop instead of checking conditionally.
    if(colId == outCols - 1) {
        width = lastWidth; // Obtain last width for the last column
    }

    // Initialize maximum value
    float currentMax = -FLT_MAX; // Use negative float max for comparison

    // Optimize: Loop unrolling
    for(int i = 0; i < width; ++i) {
        // Calculate and compare product in one step
        float product = b[i] * localMask[i];
        if(product > currentMax) {
            currentMax = product;
        }
    }

    // Write the result into the output
    out[rowId * outCols + colId] = currentMax; // Ensure correct output index calculation
}