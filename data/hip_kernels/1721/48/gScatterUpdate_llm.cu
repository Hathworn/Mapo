#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gScatterUpdate(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory to minimize global memory accesses
    __shared__ int sharedSparseIndices[1024];  // Assuming blockDim.x <= 1024
    __shared__ float sharedSparseData[1024];

    if (idx < sparseSize) {
        sharedSparseIndices[threadIdx.x] = sparseIndices[idx];
        sharedSparseData[threadIdx.x] = sparseData[idx];

        __syncthreads();

        int index = sharedSparseIndices[threadIdx.x];
        if (index >= -offset && index + offset < denseSize) {
            denseData[index + offset] = sharedSparseData[threadIdx.x];
        }
    }
}