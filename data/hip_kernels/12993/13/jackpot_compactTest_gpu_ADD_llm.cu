#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void jackpot_compactTest_gpu_ADD(uint32_t *data, uint32_t *partial_sums, int len)
{
    __shared__ uint32_t buf;
    int id = ((blockIdx.x * blockDim.x) + threadIdx.x);

    if (id >= len) return; // Correct boundary condition (>=) for global threads

    // Use a local variable to reduce global memory accesses
    int thread_id = threadIdx.x;

    if (thread_id == 0)
    {
        buf = partial_sums[blockIdx.x]; // Load partial sum into shared memory
    }

    __syncthreads(); // Ensure the partial sum is loaded before adding

    // Reduce shared memory bank conflicts by using local variable
    uint32_t temp_buf = buf; 
    data[id] += temp_buf;
}