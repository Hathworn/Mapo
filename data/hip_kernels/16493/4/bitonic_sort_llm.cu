#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitonic_sort(int* arrayIn, int* arrayOut, int arrayLen, int chunkSize) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // If within bounds
    if (idx < arrayLen) {
        int myValue = arrayIn[idx];
        int chunkStart = (idx / chunkSize) * chunkSize;
        int chunkMid = chunkStart + (chunkSize >> 1); // Use bitwise shift for division by 2
        int partnerIndex = chunkSize - (idx - chunkStart) - 1 + chunkStart;

        // Check partner index bounds
        if (partnerIndex < arrayLen) {
            int partnerValue = arrayIn[partnerIndex];
            
            // Avoid multiple conditional comparisons
            bool compare = (myValue <= partnerValue);
            int min = compare ? myValue : partnerValue;
            int max = compare ? partnerValue : myValue;
            
            myValue = (idx < chunkMid) ? min : max;
        }
        
        // Write result
        arrayOut[idx] = myValue;
    }
}