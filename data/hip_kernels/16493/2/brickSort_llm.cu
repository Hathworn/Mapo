#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void brickSort(int* array, int arrayLen, int p) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit if idx is out of bounds or not applicable for the current phase
    if (idx >= arrayLen - 1 || (idx % 2 != p % 2)) 
        return;

    // Swap elements if out of order
    if (array[idx] > array[idx + 1]) {
        int tmp = array[idx + 1];
        array[idx + 1] = array[idx];
        array[idx] = tmp;
    }
}