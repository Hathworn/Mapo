#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pi_estimation(float *pi, hiprandState *states)
{
    unsigned int threadID = threadIdx.x + blockDim.x * blockIdx.x;
    int count = 0;
    float x, y, z;

    // Initialize random state outside the loop
    hiprand_init(0, threadID, 0, &states[threadID]);

    // Use registers to store frequently used values
    hiprandState localState = states[threadID];

    for (int i = 0; i < trial_number; i++)
    {
        x = hiprand_uniform(&localState); // Obtaining random x
        y = hiprand_uniform(&localState); // Obtaining random y
        z = (x * x + y * y); // Calculate z
        if (z <= 1.0f) // Check if point is in the unit circle
        {
            count += 1;
        }
    }

    states[threadID] = localState; // Save modified state back to global memory
    pi[threadID] = 4.0f * count / (float)trial_number; // Estimate PI
}