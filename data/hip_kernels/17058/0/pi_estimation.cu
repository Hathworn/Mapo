#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pi_estimation(float *pi, hiprandState *states)
{
unsigned int threadID = threadIdx.x + blockDim.x * blockIdx.x;
int count = 0;
float x,y,z;

hiprand_init(0, threadID, 1, &states[threadID]); //(seed, sequence number, offset, hiprandState)

for(int i = 0; i < trial_number; i++)
{
x = hiprand_uniform(&states[threadID]);//return sequence number of pseudorandom uniformly distributed.
y = hiprand_uniform(&states[threadID]);
z = (x*x + y*y);
if(z <= 1.0f) //if x,y in unit circle
{
count += 1;
}
}
pi[threadID] = 4.0f * count/(float)trial_number; //estimate PI value 4*count/number of trial
}