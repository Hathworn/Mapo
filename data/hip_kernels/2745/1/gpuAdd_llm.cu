#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void gpuAdd(int d_a, int d_b, int *d_c) {
    // Use shared memory for better performance
    __shared__ int temp;
    if (threadIdx.x == 0) {
        temp = d_a + d_b;
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        *d_c = temp;
    }
}