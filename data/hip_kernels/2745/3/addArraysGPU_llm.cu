#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addArraysGPU(int* a, int* b, int* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < n) { // Ensure within bounds
        c[i] = a[i] + b[i];
    }
}