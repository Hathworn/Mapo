#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//============================================================================
// Name        : CudaMap.cu
// Author      : Hang
//============================================================================

using namespace std;

__global__ void addTen(float* d, int count) {

    // Optimize calculation of tid using simplified expressions
    int tid = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
              + threadIdx.z * blockDim.x * blockDim.y
              + threadIdx.y * blockDim.x + threadIdx.x;

    // Use early return to avoid unnecessary indexing
    if (tid >= count) return;

    d[tid] += 10; // Simplify addition operation
}