#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**
* Vector Addition - Simple addition using HIP.
* Author - Malhar Bhatt
* Subject - High Performance Computing
*/

/** Function Add -
* Usage - Add 2 values
* Returns - Void
*/
__global__ void add(int num1, int num2, int *ans)
{
    // Use shared memory to minimize global memory access
    __shared__ int temp_1, temp_2;

    // Load data into shared memory
    if (threadIdx.x == 0) {
        temp_1 = num1;
        temp_2 = num2;
    }

    // Ensure all threads have loaded data
    __syncthreads();

    // Perform operation
    if (threadIdx.x == 0) {
        *ans = temp_1 + temp_2;
    }
    // No need for further __syncthreads() as only one thread updates *ans
}