#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c) {
    // Use shared memory to minimize global memory access
    extern __shared__ int sharedMem[];
    int* sharedC = sharedMem;

    // Calculate sum using shared memory
    *sharedC = a + b;

    // Write result back to global memory
    *c = *sharedC;
}