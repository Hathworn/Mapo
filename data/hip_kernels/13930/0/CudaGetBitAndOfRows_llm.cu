#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CudaGetBitAndOfRows(unsigned int* table1D, unsigned int* row, int rowSize, int tableRowCount)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalSize = tableRowCount * rowSize;

    // Ensure within bounds and calculate once.
    if (idx < totalSize)
    {
        unsigned int rowValue = row[idx % rowSize]; // Read row element once
        table1D[idx] &= rowValue;                   // Use compound assignment
    }
}