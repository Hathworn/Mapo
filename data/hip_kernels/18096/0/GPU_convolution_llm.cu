#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 32

struct event_pair
{
    hipEvent_t start;
    hipEvent_t end;
}

__global__ void GPU_convolution(float *channel, float *mask, float *result, int dimMask, int dimW, int dimH) {
    // Cache mask to shared memory for faster access
    __shared__ float sharedMask[dimMask * dimMask];
    if (threadIdx.y * dimMask + threadIdx.x < dimMask * dimMask) {
        sharedMask[threadIdx.y * dimMask + threadIdx.x] = mask[threadIdx.y * dimMask + threadIdx.x];
    }
    __syncthreads();

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    int nidRow = Row - dimMask / 2;
    int nidCol = Col - dimMask / 2;

    int tid = Row * dimW + Col;

    // Use temporary variable to avoid repeated global memory access
    float pixelValue = 0.0f;

    // Only compute if within bounds
    if (Row < dimH && Col < dimW) {
        for (int i = 0; i < dimMask; ++i) {
            int x = nidRow + i;
            for (int j = 0; j < dimMask; ++j) {
                int y = nidCol + j;
                if (x >= 0 && y >= 0 && x < dimH && y < dimW) {
                    pixelValue += sharedMask[i * dimMask + j] * channel[x * dimW + y];
                }
            }
        }
        // Clamp final value
        result[tid] = fminf(fmaxf(pixelValue, 0.0f), 255.0f);
    }
}