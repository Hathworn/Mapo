#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_cmp_kernel(std::size_t n, int* aptr, int* bptr, int* rptr) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform comparison only if within bounds
    if (i < n) {
        int cmp = aptr[i] < bptr[i];
        
        // Use shared memory to aggregate results
        extern __shared__ int sdata[];
        sdata[threadIdx.x] = cmp;
        
        // Synchronize threads within block
        __syncthreads();

        // Reduce results within block
        if (threadIdx.x == 0) {
            int blockResult = 0;
            for (int j = 0; j < blockDim.x; j++) {
                blockResult |= sdata[j];
            }
            if (blockResult) atomicOr(rptr, 1);
        }
    }
}