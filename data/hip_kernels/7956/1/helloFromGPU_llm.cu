#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "includes.h"

/*
1. Memory Copy Cost   One-Step
2. Straggler: Ring-based
**/
using namespace std;

#define Malloc(type,n) (type *)malloc((n)*sizeof(type))

__global__ void helloFromGPU(void)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Compute global index
    if (idx == 0) // Ensure only one thread prints the message
    {
        printf("Hello from GPU\n");
    }
}
