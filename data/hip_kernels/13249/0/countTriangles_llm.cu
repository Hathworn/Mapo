#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void countTriangles(uint2 *validPoints, int *count)
{
    // Calculate unique thread index based on block indices
    int idx1 = blockIdx.x;
    int idx2 = blockIdx.y;
    int idx3 = blockIdx.z;

    // Ensure unique triangle by ensuring index order: x < y < z
    if (idx1 >= idx2 || idx2 >= idx3 || idx1 >= idx3)
        return;

    // Load valid points based on calculated indices
    uint2 x = validPoints[idx1];
    uint2 y = validPoints[idx2];
    uint2 z = validPoints[idx3];

    // Check if points are coplanar or coincident
    if ((x.x == y.x && x.x == z.x) || (x.y == y.y && x.y == z.y) ||
        (y.x == z.x && x.x == y.x) || (y.y == z.y && x.y == y.y))
        return;

    if ((x.x == y.x && x.y == y.y) || (y.x == z.x && y.y == z.y) || 
        (x.x == z.x && x.y == z.y))
        return;

    // Increment count if a valid triangle is formed
    atomicAdd(count, 1);
}