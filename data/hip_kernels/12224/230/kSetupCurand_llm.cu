#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate unique thread ID and ensure threads are not wasting resources
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x; 

    if (tidx < NUM_RND_THREADS_PER_BLOCK * gridDim.x) {
        /* Each thread gets same seed, a different sequence number, no offset */
        hiprand_init(seed, tidx, 0, &state[tidx]);
    }
}