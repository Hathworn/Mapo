#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0.0f;

    // Improved loop iteration to reduce divergence
    for (uint i = eidx; i < numElements; i += blockDim.x * gridDim.x) {
        shmem[threadIdx.x] += a[i] * b[i];
    }

    __syncthreads();

    // Optimized reduction using fewer synchronization points
    for (uint s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            shmem[threadIdx.x] += shmem[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Single thread writes the block's result to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}