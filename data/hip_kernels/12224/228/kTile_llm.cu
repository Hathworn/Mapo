#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Compute global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total number of threads
    const int numThreads = blockDim.x * gridDim.x;

    // Use precomputed variables for repeated calculations
    const uint srcSize = srcWidth * srcHeight;
    const uint tgtSize = tgtWidth * tgtHeight;

    for (uint i = idx; i < tgtSize; i += numThreads) {
        // Compute y and x coordinates for target
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        // Map target coordinates to source coordinates
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Efficiently copy from source to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}