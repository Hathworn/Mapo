#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for better memory coalescing and parallel efficiency
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  
    
    // Use one thread to perform the division to prevent race conditions
    if (idx == 0) {
        *accuracy /= N;
    }
}