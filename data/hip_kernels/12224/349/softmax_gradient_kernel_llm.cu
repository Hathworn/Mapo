#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    
    // Use warp shuffle for efficient reduction within a block
    float tmp = 0;
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    
    // Perform reduction across the block using shared memory
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Reduce using binary tree reduction for better global synchronization
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (idx < offset) {
            reduction_buffer[idx] += reduction_buffer[idx + offset];
        }
        __syncthreads();
    }
    
    // Compute gradient after obtaining the complete inner product
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}