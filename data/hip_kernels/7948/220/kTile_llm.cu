#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate global index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    // Loop through each target element that this thread is responsible for
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        // Compute source indices using modulo arithmetic
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Read from source and write to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}