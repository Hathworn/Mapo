#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Calculate probability of correct case if the most-probable label is equal to the true label
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;

            // Use a single loop to find `numMax`
            #pragma unroll
            for (int i = 0; i < numOut; i++) {
                numMax += (__float2int_rd(probs[i * numCases + tx] == maxp));
            }

            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}