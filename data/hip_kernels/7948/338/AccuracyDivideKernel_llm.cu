#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel using block and thread indexing
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure only one thread performs the division
    if (idx == 0) {
        *accuracy /= N;
    }
}