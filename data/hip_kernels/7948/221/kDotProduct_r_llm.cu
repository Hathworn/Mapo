#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;  // Reduce shared memory access by using a local variable

    if (eidx < numElements) {
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            sum += a[eidx] * b[eidx];
        }
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Reduce within shared memory
    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    __syncthreads();

    // Use warp-level primitives to reduce further
    if (threadIdx.x < 32) {
        volatile float* myshmem = shmem;
        myshmem[threadIdx.x] += myshmem[threadIdx.x + 32];
        myshmem[threadIdx.x] += myshmem[threadIdx.x + 16];
        myshmem[threadIdx.x] += myshmem[threadIdx.x + 8];
        myshmem[threadIdx.x] += myshmem[threadIdx.x + 4];
        myshmem[threadIdx.x] += myshmem[threadIdx.x + 2];
        myshmem[threadIdx.x] += myshmem[threadIdx.x + 1];
    }

    // Write the block's result to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}