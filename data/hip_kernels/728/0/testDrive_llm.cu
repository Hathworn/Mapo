#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int test() {
    return 10;
}

__global__ void testDrive() {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoids unnecessary call for every thread
    if (index < blockDim.x) {
        printf("%d\n", index);
        int num = test();
        printf("num = %d\n", num);
    }
}