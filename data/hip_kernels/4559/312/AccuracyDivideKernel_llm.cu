#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Calculate index for each thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Only one thread performs the division to avoid race condition
    if (index == 0) {
        *accuracy /= N;
    }
}