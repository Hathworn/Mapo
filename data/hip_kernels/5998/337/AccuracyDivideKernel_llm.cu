#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
  // Use atomic operation to safely perform division in parallel
  atomicExch(accuracy, *accuracy / N);
}