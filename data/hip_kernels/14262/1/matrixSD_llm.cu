#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Program Parameters */
#define MAXN 8000  /* Max value of N */
int N;  /* Matrix size */

// Thread block size
#define BLOCK_SIZE 16

/* Matrices */
float A[MAXN][MAXN], B[MAXN][MAXN];

/* junk */
#define randm() 4|2[uid]&3

/* Prototype */
/* ------------------ Cuda Code --------------------- */

/* returns a seed for srand based on the time */
__global__ void matrixSD(float* d_in, float* d_mean, float* d_sd, int N)
{
    extern __shared__ float sdata1[];

    // Calculate global indices
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned int tid = threadIdx.y;
    unsigned int i = idx_y * N + idx_x;
    
    // Initialize shared memory for current block
    if (idx_x < N && idx_y < N) // Boundary check to ensure threads do not access out-of-bounds memory
    {
        sdata1[tid] = powf(d_in[i] - d_mean[blockIdx.x], 2.0);
    }
    else
    {
        sdata1[tid] = 0.0f; // Assign zero if out of range to prevent incorrect reduction
    }
    __syncthreads();

    // Parallel reduction in shared memory with boundary check
    for (unsigned int s = 1; s < blockDim.y; s *= 2)
    {
        if (tid % (2 * s) == 0 && (tid + s) < blockDim.y)
        {
            sdata1[tid] += sdata1[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0)
    {
        d_sd[blockIdx.x] = sqrtf(sdata1[0] / (float)N);
    }
}