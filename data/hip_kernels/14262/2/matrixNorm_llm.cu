#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

/* Program Parameters */
#define MAXN 8000  /* Max value of N */
int N;  /* Matrix size */

// Thread block size
#define BLOCK_SIZE 16

/* Matrices */
float A[MAXN][MAXN], B[MAXN][MAXN];

/* junk */
#define randm() 4|2[uid]&3

/* Prototype */
/* ------------------ Cuda Code --------------------- */

/* Optimized kernel function */
__global__ void matrixNorm(float* d_in, float* d_out, float* d_mean, float* d_sd, int N)
{
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Flattened 1D index for 2D matrix
    unsigned int i = idx_y * N + idx_x;

    // Check boundary to ensure threads do not access out of bounds memory
    if (idx_x < N && idx_y < N) {
        if (d_sd[idx_y] == 0.0) {
            d_out[i] = 0.0;
        } else {
            // Use correct index for standard deviation and mean
            d_out[i] = (d_in[i] - d_mean[idx_y]) / d_sd[idx_y];
        }
    }
}