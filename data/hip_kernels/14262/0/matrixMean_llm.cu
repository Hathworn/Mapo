#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Program Parameters */
#define MAXN 8000  /* Max value of N */
int N;  /* Matrix size */

// Thread block size
#define BLOCK_SIZE 16

/* Matrices */
float A[MAXN][MAXN], B[MAXN][MAXN];

/* junk */
#define randm() 4|2[uid]&3

/* ------------------ Cuda Code --------------------- */

__global__ void matrixMean(float* d_in, float* d_mean, int N)
{
    extern __shared__ float sdata[];

    // Calculate unique thread index
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned int tid = threadIdx.x + threadIdx.y * blockDim.x;
    unsigned int gridSize = blockDim.x * gridDim.x;

    // Initialize shared memory
    float sum = 0.0f;
    while (idx_y * N + idx_x < N * N)
    {
        sum += d_in[idx_y * N + idx_x];
        idx_x += gridSize;
    }
    sdata[tid] = sum;
    __syncthreads();

    // Reduction in shared memory
    for (unsigned int s=blockDim.x * blockDim.y / 2; s>0; s>>=1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0)
    {
        d_mean[blockIdx.y * gridDim.x + blockIdx.x] = sdata[0] / (float)(N * N);
    }
}