#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    // Calculate global index
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index is within bounds
    if (idx < N) {
        output[blockIdx.x * N + idx] = -col_sum[idx];
    }
}