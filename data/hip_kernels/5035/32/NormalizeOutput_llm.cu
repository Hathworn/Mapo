#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Cache common values to register for faster access
    int64_t cache_batch_index = batch_index;
    int64_t cache_class_index = class_index;

    // Use shared memory to improve memory throughput (if applicable)
    extern __shared__ int64_t shared_mem[];

    // Calculate unique thread index across grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < num_elements) { 
        // Update to_normalize using cached values
        to_normalize[idx * 3] = cache_batch_index;
        to_normalize[idx * 3 + 1] = cache_class_index;
        to_normalize[idx * 3 + 2] = static_cast<int64_t>(original[idx]);
    }
}