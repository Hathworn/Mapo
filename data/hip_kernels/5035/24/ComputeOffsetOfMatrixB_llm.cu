#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Calculate the unique index for each thread
    int32_t index = blockIdx.x * N + threadIdx.x;
    // Use a stride loop to process all elements
    for (int32_t i = index; i < N * (blockIdx.x + 1); i += blockDim.x) {
        output[i] = -row_sum[blockIdx.x];
    }
}