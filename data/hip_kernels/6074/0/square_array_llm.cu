#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square_array(float *a, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Using loop unrolling for optimization.
    if (idx < N) 
    {
        a[idx] = a[idx] * a[idx];
        if (idx + blockDim.x < N) a[idx + blockDim.x] = a[idx + blockDim.x] * a[idx + blockDim.x];
        if (idx + 2 * blockDim.x < N) a[idx + 2 * blockDim.x] = a[idx + 2 * blockDim.x] * a[idx + 2 * blockDim.x];
        if (idx + 3 * blockDim.x < N) a[idx + 3 * blockDim.x] = a[idx + 3 * blockDim.x] * a[idx + 3 * blockDim.x];
    }
}