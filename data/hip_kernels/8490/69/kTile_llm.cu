#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    
    // Optimize loop using shared memory to reduce global memory access
    extern __shared__ float sharedSrc[];
    
    const uint srcSize = srcWidth * srcHeight;
    for (uint i = idx; i < srcSize; i += numThreads) {
        sharedSrc[i] = src[i];
    }
    __syncthreads();
    
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        const uint srcIdx = srcY * srcWidth + srcX;
        
        // Use shared memory to access the source element
        tgt[i] = sharedSrc[srcIdx];
    }
}