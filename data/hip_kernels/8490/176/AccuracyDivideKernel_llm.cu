#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel to ensure parallel execution
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Calculate the index position for the current thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure only one thread performs division to avoid race condition
    if (idx == 0) {
        *accuracy /= N;
    }
}