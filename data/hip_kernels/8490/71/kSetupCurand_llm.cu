#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Optimize: Use a loop to initialize multiple states per thread if needed
    for (int i = 0; i < NUM_RND_THREADS_PER_BLOCK; i += blockDim.x){
        int index = tidx + i;
        if (index < NUM_RND_THREADS_PER_BLOCK * gridDim.x) {
            hiprand_init(seed, index, 0, &state[index]);
        }
    }
}