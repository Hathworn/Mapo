#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0; // Use a register to accumulate partial sums

    if (eidx < gridDim.x * DP_BLOCKSIZE) {
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            sum += a[eidx] * b[eidx]; // Accumulate in register
        }
    }
    shmem[threadIdx.x] = sum; // Store the result in shared memory
    __syncthreads();

    // Efficient parallel reduction in shared memory
    for (uint stride = DP_BLOCKSIZE / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}