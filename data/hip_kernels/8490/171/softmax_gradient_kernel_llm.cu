#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    const int idx = threadIdx.x;
    float tmp = 0.0f;

    // Pointer offsets for each block
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;

    // Compute partial sums for the inner product using stride loop
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Single thread reduction to accumulate final result
    if (idx == 0) {
        for (int i = 1; i < blockDim.x; ++i) {
            reduction_buffer[0] += reduction_buffer[i];
        }
    }
    __syncthreads();

    // Apply the gradient calculation using computed inner product
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}