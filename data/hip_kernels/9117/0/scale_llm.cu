#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

float *A,*L,*U,*input;
void arrayInit(int n);
void verifyLU(int n);
void updateLU(int n);
void freemem(int n);

/*
*/

__global__ void scale(float *a, int b, int c) {
    // Use thread index to perform parallel division calculations
    int index = c;
    int size = b;
    int k = index + 1 + threadIdx.x; // Offset by threadIdx.x

    for (; k < size; k += blockDim.x) { // Increment by blockDim.x for parallel iteration
        a[size * index + k] = a[size * index + k] / a[size * index + index];
    }
}