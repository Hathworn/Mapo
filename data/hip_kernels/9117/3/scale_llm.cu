#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale(float *a, int size, int c) {
    int index = c;
    // Using blockIdx and threadIdx for parallel processing
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure tid is within bounds and processing only necessary elements
    for (int k = index + 1 + tid; k < size; k += blockDim.x * gridDim.x) {
        a[size * index + k] = (float) a[size * index + k] / a[size * index + index];
    }
}