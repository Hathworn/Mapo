#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

float *A, *L, *U, *input;
void arrayInit(int n);
void verifyLU(int n);
void updateLU(int n);
void freemem(int n);

__global__ void reduce(float *a, int size, int c) {
    int tid = blockIdx.x;  // Use blockIdx for row identification
    int thid = threadIdx.x; // Use threadIdx.x for column offset

    // Calculate global column index for this thread
    int global_j = c + 1 + thid; 

    // Process columns in this block with stride equal to blockDim.x
    for (; global_j < size; global_j += blockDim.x) {
        int a_idx1 = (tid + c + 1) * size + global_j;
        int a_idx2 = (tid + c + 1) * size + c;
        int a_idx3 = c * size + global_j;
        
        // Apply the reduction formula for LU decomposition
        a[a_idx1] -= a[a_idx2] * a[a_idx3];
    }
}