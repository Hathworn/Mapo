#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global thread index
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within image boundaries
    if (i < h && j < w) 
    {
        // Compute position index
        const int pos = i * s + j;

        // Load normalization factor and compute inverse
        float scale = normalization_factor[pos];
        float invScale = (scale == 0.0f) ? 1.0f : __fdividef(1.0f, scale); // Fast reciprocal

        // Normalize the image
        image[pos] *= invScale;
    }
}