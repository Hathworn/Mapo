#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate global index for the thread
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread operates only on valid data points
    if (ptidx < npoints)
    {
        // Use register storage for the local variable to optimize memory usage
        register short2 loc = loc_[ptidx];

        // Multiply once and store in separate arrays
        float mul_scale = loc.x * scale;
        x[ptidx] = mul_scale;

        mul_scale = loc.y * scale;
        y[ptidx] = mul_scale;
    }
}