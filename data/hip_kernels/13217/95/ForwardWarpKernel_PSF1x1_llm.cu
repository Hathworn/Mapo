#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Calculate row and column indices for the current thread
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Return if the thread is outside the image bounds
    if (i >= h || j >= w) return;

    // Pre-calculate offsets for flow and image arrays
    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Fetch flow values from global memory
    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Calculate the bottom left corner of the target pixel using flow displacement
    float cx = u_ * time_scale + (float)j + 1.0f;
    float cy = v_ * time_scale + (float)i + 1.0f;

    // Determine pixel coordinates for the target location
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Fast path to check if target pixel is within bounds
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        float value = src[image_row_offset + j];

        // Atomically add the value to the target pixel in destination image
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}