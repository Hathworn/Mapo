#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Early exit for threads outside image bounds
    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Compute the bottom-left corner of the target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    
    // Calculate fractional and integer parts
    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);

    // Target pixel integer coordinates
    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];
    float weight;

    // Bottom-right corner
    if (tx < w && tx >= 0 && ty < h && ty >= 0)
    {
        weight = dx * dy;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Bottom-left corner
    if (--tx >= 0)
    {
        if (tx < w && ty < h && ty >= 0)
        {
            weight = (1.0f - dx) * dy;
            _atomicAdd(dst + ty * image_stride + tx, value * weight);
            _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
        }
    }

    // Top-left corner
    if (--ty >= 0)
    {
        if (tx < w && tx >= 0)
        {
            weight = (1.0f - dx) * (1.0f - dy);
            _atomicAdd(dst + ty * image_stride + tx, value * weight);
            _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
        }
    }

    // Top-right corner
    if (++tx < w)
    {
        if (ty < h && ty >= 0)
        {
            weight = dx * (1.0f - dy);
            _atomicAdd(dst + ty * image_stride + tx, value * weight);
            _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
        }
    }
}