#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void reduce(int *g_idata, int l1, int l2) {
    extern __shared__ int sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + tid;

    // Load data into shared memory
    sdata[tid] = (i < l1) ? g_idata[i] : 0;
    __syncthreads();

    // Parallel Reduction (l2 must be power of 2)
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global mem
    if (tid == 0) {
        g_idata[blockIdx.x] = sdata[0];
    }
}