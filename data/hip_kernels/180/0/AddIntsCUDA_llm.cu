#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AddIntsCUDA(int *a, int *b) //Kernel Definition
{
    // Calculate global index for the single thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform addition only if index is 0 to ensure the operation is performed once
    if (idx == 0)
    {
        *a = *a + *b;
    }
}