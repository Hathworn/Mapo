#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 24
#define MAXPOOL_INPUT_FILENAME "input.txt"
#define A_FILENAME "a.txt"
#define B_FILENAME "b.txt"
#define C_FILENAME "c.txt"

using namespace std;

__global__ void gemm(float *a, float *b, float *c, const float alpha, const float beta, float *output, const int input_size){
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    if(row >= input_size || col >= input_size) { return; }

    __shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
    __shared__ float s_b[TILE_WIDTH][TILE_WIDTH];

    float result = 0;

    for(int p = 0; p < input_size/TILE_WIDTH + 1; p++){
        int a_index = row * input_size + p * TILE_WIDTH + tx;
        int b_index = (p * TILE_WIDTH + ty) * input_size + col;

        // Load tiles into shared memory
        s_a[ty][tx] = (a_index < input_size * input_size) ? a[a_index] : 0;
        s_b[ty][tx] = (b_index < input_size * input_size) ? b[b_index] : 0;

        __syncthreads(); // Synchronize to ensure tiles are loaded

        // Compute partial matrix product
        for(int i = 0; i < TILE_WIDTH; i++) {
            result += s_a[ty][i] * s_b[i][tx];
        }

        __syncthreads(); // Synchronize before loading new tiles
    }

    if(row < input_size && col < input_size) {
        // Write result to output matrix
        output[row * input_size + col] = (alpha * result) + (beta * c[row * input_size + col]);
    }
}