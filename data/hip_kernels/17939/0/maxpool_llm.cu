#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 24 // set TILE_WIDTH 16 for the evaluation!
#define MAXPOOL_INPUT_FILENAME "input.txt"
#define A_FILENAME "a.txt"
#define B_FILENAME "b.txt"
#define C_FILENAME "c.txt"

using namespace std;

__global__ void maxpool(float *input, float *output, const int input_size, const int filter_size) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    // Boundary check
    if (col >= input_size / filter_size || row >= input_size / filter_size) return;

    float Max = -FLT_MAX; // Use float max negative value for max comparison

    // Iterate over filter size
    for (int i = 0; i < filter_size; i++) {
        for (int j = 0; j < filter_size; j++) {
            int inputRow = filter_size * row + i;
            int inputCol = filter_size * col + j;
            if (inputRow < input_size && inputCol < input_size) {
                float value = input[inputRow * input_size + inputCol]; // Calculate index for vectorized matrix
                Max = max(Max, value); // Use built-in max function
            }
        }
    }

    output[(input_size / filter_size) * row + col] = Max; // Assign calculated max to the output
}