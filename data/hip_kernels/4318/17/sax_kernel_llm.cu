#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sax_kernel(const float a, const float* __restrict__ x, float* __restrict__ result, unsigned int len) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        result[idx] = a * x[idx]; // Use restrict for improved memory access
    }
}