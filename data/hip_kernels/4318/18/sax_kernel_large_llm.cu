#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sax_kernel_large(const float a, const float* x, float* result, unsigned int len, unsigned int rowsz) {
    // Calculate the global index for each thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * rowsz;

    // Ensure valid index before accessing memory
    if (idx < len) {
        // Efficiently compute the result
        result[idx] = a * x[idx];
    }
}