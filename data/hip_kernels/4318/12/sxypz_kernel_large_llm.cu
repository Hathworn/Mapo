#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sxypz_kernel_large(float a, const float* x, const float* y, const float* z, float* result, unsigned int len, unsigned int rowsz) {
    // Calculate a unique 1D index within the grid
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * rowsz;

    // Ensure all threads with valid indices compute the result
    if (idx < len) {
        // Load values before computing result to reduce global memory access
        float x_val = x[idx];
        float y_val = y[idx];
        float z_val = z[idx];
        
        // Compute and store the result
        result[idx] = a * x_val * y_val + z_val;
    }
}