#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rsqrt_kernel_large(float* x, unsigned int len, unsigned int rowsz) {
    // Compute global index
    unsigned int idx = blockIdx.y * rowsz + blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if idx is within bounds and perform operation
    if (idx < len && x[idx] > 0) {
        x[idx] = rsqrtf(x[idx]);  // Use rsqrtf for better performance with float
    }
}