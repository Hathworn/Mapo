#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for computing square root
__global__ void sqrt_kernel_large(float* x, unsigned int len, unsigned int rowsz) {
    // Cache thread and block indices
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int bj = blockIdx.y;

    // Compute global index with improved readability
    unsigned int idx = tid + bid * blockDim.x + bj * rowsz;
    
    // Check bounds once
    if (idx < len) {
        // Compute square root
        x[idx] = sqrt(x[idx]);
    }
}