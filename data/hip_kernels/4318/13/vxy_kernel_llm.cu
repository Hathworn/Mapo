#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vxy_kernel(const float* __restrict__ x, const float* __restrict__ y, float* __restrict__ result, unsigned int len) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure memory access is coalesced and avoid redundant calculations
    if (idx < len) {
        result[idx] = x[idx] * y[idx];
    }
}