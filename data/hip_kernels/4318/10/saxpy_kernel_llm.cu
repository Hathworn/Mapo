#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy_kernel(const float a, const float* __restrict__ x, const float* __restrict__ y, float* __restrict__ result, unsigned int len) {
    // Calculate global thread index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform computation only for valid indices
    if (idx < len) {
        // Cache load for x and y to reduce global memory access times
        float x_val = x[idx];
        float y_val = y[idx];

        // Store the result back at the same index
        result[idx] = a * x_val + y_val;
    }
}