#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy_kernel_large(const float a, const float* x, const float* y, float* result, unsigned int len, unsigned int rowsz) {
    // Calculate index with grid stride loop
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * rowsz;
    unsigned int totalThreads = gridDim.x * blockDim.x;

    for (unsigned int i = idx; i < len; i += totalThreads) {
        result[i] = a * x[i] + y[i];
    }
}