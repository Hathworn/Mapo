#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vxy_kernel_large(const float* x, float* y, float* result, unsigned int len, unsigned int rowsz) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x + rowsz * blockIdx.y;
    
    // Unroll the loop for better performance
    if (idx < len) {
        result[idx] = x[idx] * y[idx];
        if (idx + blockDim.x < len) result[idx + blockDim.x] = x[idx + blockDim.x] * y[idx + blockDim.x];
    }
}