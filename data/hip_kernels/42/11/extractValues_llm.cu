#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void extractValues(void* fb, int* voxels, int num_voxels, int* values) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory to reduce global memory accesses
    extern __shared__ int shared_voxels[];
    
    if (index < num_voxels) {
        // Load voxels into shared memory
        shared_voxels[threadIdx.x] = voxels[index];
        __syncthreads();

        // Process values from shared memory
        float* tile = (float*)fb;
        values[index] = __float_as_int(tile[shared_voxels[threadIdx.x]]);
    }
}