#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void splitNodes(int* octree, int* numNodes, int poolSize, int startNode) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds to avoid unnecessary computations
    if (index < poolSize) {
        int node = octree[2 * (index + startNode)];

        // Split node if flagged
        if (node & 0x40000000) {
            // Atomically add 8 to get new node index
            int newNode = atomicAdd(numNodes, 8);

            // Update node to point to new node tile
            int baseIdx = 2 * (index + startNode);
            octree[baseIdx] = (node & 0xC0000000) | (newNode & 0x3FFFFFFF);

            // Use unrolled loop for initialization of new child nodes
            #pragma unroll
            for (int off = 0; off < 8; ++off) {
                int childIdx = 2 * (newNode + off);
                octree[childIdx] = 0;
                octree[childIdx + 1] = 0;
            }
        }
    }
}