#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_pow_grad_device(int *x, int power, int *grad, int *out, bool grad_is_scalar, unsigned int size) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    // Loop unrolling to enhance performance
    for (unsigned int i = idx; i < size; i += stride * 4) {
        if (i < size) {
            float x_val0 = (float) x[i];
            out[i] = grad[(grad_is_scalar) ? 0 : i] * power * (int) powf(x_val0, power - 1);
        }
        if (i + stride < size) {
            float x_val1 = (float) x[i + stride];
            out[i + stride] = grad[(grad_is_scalar) ? 0 : i + stride] * power * (int) powf(x_val1, power - 1);
        }
        if (i + stride * 2 < size) {
            float x_val2 = (float) x[i + stride * 2];
            out[i + stride * 2] = grad[(grad_is_scalar) ? 0 : i + stride * 2] * power * (int) powf(x_val2, power - 1);
        }
        if (i + stride * 3 < size) {
            float x_val3 = (float) x[i + stride * 3];
            out[i + stride * 3] = grad[(grad_is_scalar) ? 0 : i + stride * 3] * power * (int) powf(x_val3, power - 1);
        }
    }
}