#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_log_full_device(int *x, int *out, unsigned int size, int epsilon) {

    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    // Unroll the loop to improve performance
    for (unsigned int i = idx; i < size; i += stride * 4) {
        if (i < size) {
            out[i] = (int) logf((float) x[i] + epsilon);
        }
        if (i + stride < size) {
            out[i + stride] = (int) logf((float) x[i + stride] + epsilon);
        }
        if (i + 2 * stride < size) {
            out[i + 2 * stride] = (int) logf((float) x[i + 2 * stride] + epsilon);
        }
        if (i + 3 * stride < size) {
            out[i + 3 * stride] = (int) logf((float) x[i + 3 * stride] + epsilon);
        }
    }
}