#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_sigmoid_full_device(unsigned int size, int *x, int *out) {
    // Calculate the global thread index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process all elements within stride
    while (idx < size) {
        // Compute sigmoid function and store the result
        out[idx] = 1 / (1 + abs(x[idx]));
        // Move to the next element in stride
        idx += blockDim.x * gridDim.x;
    }
}