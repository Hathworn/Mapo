#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Compute target pixel coordinates
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // Extract integer part and fractional part
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    // Cast integer coordinates
    int tx = (int)px;
    int ty = (int)py;

    // Fetch source pixel value
    float value = src[image_row_offset + j];

    // Precompute weights for different pixels
    float wbr = dx * dy;
    float wbl = (1.0f - dx) * dy;
    float wul = (1.0f - dx) * (1.0f - dy);
    float wur = dx * (1.0f - dy);
    
    // Macro to perform atomic add if within bounds
    #define ATOMIC_ADD_CONDITIONALLY(tx, ty, weight) \
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0))) { \
        _atomicAdd(dst + ty * image_stride + tx, value * weight); \
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight); \
    }

    // Apply precomputed weights to relevant pixels
    ATOMIC_ADD_CONDITIONALLY(tx, ty, wbr); // Bottom right
    ATOMIC_ADD_CONDITIONALLY(tx - 1, ty, wbl); // Bottom left
    ATOMIC_ADD_CONDITIONALLY(tx - 1, ty - 1, wul); // Upper left
    ATOMIC_ADD_CONDITIONALLY(tx, ty - 1, wur); // Upper right

    #undef ATOMIC_ADD_CONDITIONALLY
}