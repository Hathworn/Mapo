#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate linear thread index globally
    int globalIndex = blockIdx.y * blockDim.y * w + threadIdx.y * w + blockIdx.x * blockDim.x + threadIdx.x;
    
    if (globalIndex >= w * h) return;
    
    // Set value using linear thread index
    image[globalIndex] = value;
}