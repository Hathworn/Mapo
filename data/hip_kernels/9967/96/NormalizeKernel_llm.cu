#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Use shared memory to reduce global memory access
    __shared__ float shared_scale[32][32]; // Adjust size based on block dimensions

    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    if (i >= h || j >= w) return;

    const int pos = i * s + j;

    // Load normalization factor into shared memory
    shared_scale[threadIdx.y][threadIdx.x] = normalization_factor[pos];
    __syncthreads(); // Synchronize shared memory access

    float scale = shared_scale[threadIdx.y][threadIdx.x];

    float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

    image[pos] *= invScale;
}