#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate thread index for each point
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index is within valid range
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];

        // Use efficient memory fetch
        float scaled_x = __half2float(loc.x) * scale;
        float scaled_y = __half2float(loc.y) * scale;

        // Write results back to global memory
        x[ptidx] = scaled_x;
        y[ptidx] = scaled_y;
    }
}