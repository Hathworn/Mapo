#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TB 128
#define GS(x) (((x) - 1) / TB + 1)

__global__ void mul_(float *input, float factor, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory for improved memory access speed
    __shared__ float shared_input[TB];

    if (id < size) {
        // Load data into shared memory
        shared_input[threadIdx.x] = input[id];
        __syncthreads();

        // Perform computation using shared memory
        shared_input[threadIdx.x] *= factor;
        __syncthreads();

        // Write back to global memory
        input[id] = shared_input[threadIdx.x];
    }
}