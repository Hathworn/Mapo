#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TB 128
#define GS(x) (((x) - 1) / TB + 1)

__global__ void fill_(float *input, float value, int size)
{
    // Use vectorized memory access for better efficiency
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop through all elements assigned to the thread
    for (int idx = id; idx < size; idx += stride) {
        input[idx] = value;
    }
}