#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define TB 128
#define GS(x) (((x) - 1) / TB + 1)

__global__ void add_(float *input, float value, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unrolling the loop for better performance
    for (int i = id; i < size; i += blockDim.x * gridDim.x) {
        input[i] = input[i] + value;
    }
}