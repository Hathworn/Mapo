#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TB 128
#define GS(x) (((x) - 1) / TB + 1)

__global__ void Normalize_get_norm_(float *input, float *norm, int size1, int size23, int size023)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size023) {
        int dim23 = id % size23;
        int dim0 = id / size23;
        
        float sum = 0.0f;
        // Use float4 vectorization if possible for faster memory access
        for (int dim1 = 0; dim1 < size1; dim1++) {
            int index = (dim0 * size1 + dim1) * size23 + dim23;
            float4 data = reinterpret_cast<float4*>(input)[index >> 2];
            sum += data.x * data.x + data.y * data.y + data.z * data.z + data.w * data.w;
        }
        norm[dim0 * size23 + dim23] = sum + 1.3e-37f;
    }
}