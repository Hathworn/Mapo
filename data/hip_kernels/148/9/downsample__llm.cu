#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TB 128
#define GS(x) (((x) - 1) / TB + 1)

__global__ void downsample_(float *input, float *output, int factor, int size3, int size)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to optimize atomic operations
    __shared__ float shared_output[TB];
    int tid = threadIdx.x;
    shared_output[tid] = 0.0f;

    if (id < size) {
        int dim3 = id % size3;
        int dim2 = id / size3;
        // Accumulate in shared memory to reduce atomic contention
        atomicAdd(&shared_output[tid], input[id] / (factor * factor));
    }

    __syncthreads();

    // Flush results from shared memory to output using atomicAdd
    if (tid == 0) {
        for (int i = 0; i < TB && (blockIdx.x * TB + i) < size; i++) {
            int idx = blockIdx.x * TB + i;
            int dim3 = idx % size3;
            int dim2 = idx / size3;
            atomicAdd(output + ((dim2 / factor) * (size3 / factor) + (dim3 / factor)), shared_output[i]);
        }
    }
}