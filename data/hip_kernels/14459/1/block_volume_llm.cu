#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

typedef unsigned int  uint;
typedef unsigned char uchar;
typedef unsigned char VolumeType;

texture<VolumeType, 3, hipReadModeNormalizedFloat> tex;
texture<VolumeType, 3, hipReadModeNormalizedFloat> tex_block;
texture<float4,  1, hipReadModeElementType> texture_float_1D;

struct Ray
{
    float3 o;   // origin
    float3 d;   // direction
};

__device__ unsigned char myMAX(unsigned char a, unsigned char b)
{
    return (a >= b) ? a : b;
}

__global__ void block_volume(unsigned char* image_p, unsigned char* dest_p, int srcWidth, int srcHeight, int srcDepth, int desWidth, int desHeight, int desDepth)
{
    unsigned int tx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int ty = blockIdx.y * blockDim.y + threadIdx.y;

    if (tx >= desWidth || ty >= desHeight) return;

    for (int i = 0; i < desDepth; i++)
    {
        unsigned char tempmax = 0;

        // Optimize memory access and loop to remove unnecessary operations
        for (int z = i * 4; z < i * 4 + 4; z++)
        {
            if (z >= srcDepth) break;
            for (int y = ty * 4; y < ty * 4 + 4; y++)
            {
                if (y >= srcHeight) break;
                for (int x = tx * 4; x < tx * 4 + 4; x++)
                {
                    if (x >= srcWidth) break;
                    tempmax = myMAX(tempmax, image_p[z * srcWidth * srcHeight + y * srcHeight + x]);
                }
            }
        }
        dest_p[i * desWidth * desHeight + ty * desWidth + tx] = tempmax; // Corrected index calculation
    }
}