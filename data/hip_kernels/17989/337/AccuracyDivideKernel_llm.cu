#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load the value into a register for faster access
    float acc_val = *accuracy; 
    // Perform the division
    acc_val /= N; 
    // Store the result back in global memory
    *accuracy = acc_val; 
}