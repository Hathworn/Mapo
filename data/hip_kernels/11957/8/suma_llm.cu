#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void suma(int a, int b, int *c){
    // Use a single thread for the operation to reduce unnecessary warp execution
    if (threadIdx.x == 0) {
        *c = a + b;
    }
}