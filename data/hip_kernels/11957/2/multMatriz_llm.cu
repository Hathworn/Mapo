#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multMatriz(float *da, float *db, float *dc, int num) {
    // Calculate row and column for the current element
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < num && j < num) {  // Ensure within bounds
        float sum = 0.0f;  // Initialize sum
        for (unsigned int k = 0; k < num; k++) {
            // Accumulate the product
            sum += da[i * num + k] * db[k * num + j];
        }
        // Store the computed value
        dc[i * num + j] = sum;
    }
}