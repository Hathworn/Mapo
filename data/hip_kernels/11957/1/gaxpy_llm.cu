#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpy(double *y, double *a, double *x, int m, int n) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];

    if (bid < m && tid < n) {  // Combined condition checks
        double product = a[bid * n + tid] * x[tid];  // Use a register for temporary storage

        // Parallel reduction for dot product
        dots_s[tid] = product;
        __syncthreads();

        for (int stride = n / 2; stride > 0; stride /= 2) {
            if (tid < stride) {
                dots_s[tid] += dots_s[tid + stride];
            }
            __syncthreads();
        }

        if (tid == 0) {
            y[bid] = dots_s[0];  // Write result to global memory
        }
    }
}