#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void indices() {
    // Calculate the unique thread ID for indexing
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Optimize the loop by removing unnecessary __syncthreads(); it's a simple printf operation
    printf("blockdimy: %d  threadx: %d  Blockidx: %d  blockdimx: %d id:  %d raiz: %f\n",
           blockDim.y, threadIdx.x, blockIdx.x, blockDim.x, id, sqrt((double)id));
}