#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__global__ void MatrixMul(float *Md, float *Nd, float *Pd, const int WIDTH) {
    // Shared memory for tiles of Md and Nd
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // Calculate the row and column index of the Pd element to work on
    unsigned int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    unsigned int row = blockIdx.y * TILE_WIDTH + threadIdx.y;

    float Pvalue = 0.0f;  // Declare the accumulator outside the loop

    // Loop over the Md and Nd tiles required to compute Pd element
    for (int m = 0; m < (WIDTH + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
        // Collaborative loading of Md and Nd tiles into shared memory
        if (row < WIDTH && (m * TILE_WIDTH + threadIdx.x) < WIDTH)
            Mds[threadIdx.y][threadIdx.x] = Md[row * WIDTH + (m * TILE_WIDTH + threadIdx.x)];
        else
            Mds[threadIdx.y][threadIdx.x] = 0.0f;  // Handle boundary

        if (col < WIDTH && (m * TILE_WIDTH + threadIdx.y) < WIDTH)
            Nds[threadIdx.y][threadIdx.x] = Nd[(m * TILE_WIDTH + threadIdx.y) * WIDTH + col];
        else
            Nds[threadIdx.y][threadIdx.x] = 0.0f;  // Handle boundary

        __syncthreads();  // Synchronize to ensure tiles are loaded

        for (int k = 0; k < TILE_WIDTH; ++k) {
            Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
        }
        __syncthreads();  // Synchronize before loading new tiles
    }

    if (row < WIDTH && col < WIDTH)
        Pd[row * WIDTH + col] = Pvalue;  // Save the result to Pd
}