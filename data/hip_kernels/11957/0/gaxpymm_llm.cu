#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpymm(double *y, double *a, double *b, int m, int n, int p) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double sharedMem[];

    if (bid < m) {
        // Load shared memory with partial results for tid < n
        if (tid < n) {
            for (int c = 0; c < p; c++) {
                sharedMem[tid * p + c] = a[bid * n + tid] * b[tid * p + c];
            }
        }
        
        __syncthreads();
        
        // Reduce within block
        if (tid == 0) {
            for (int c = 0; c < p; c++) {
                double sum = sharedMem[c];
                for (int i = 1; i < n; i++) {
                    sum += sharedMem[i * p + c];
                }
                y[bid * p + c] = sum;
            }
        }
    }
}