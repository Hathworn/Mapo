#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulSh(float *Md, float *Nd, float *Pd, const int WIDTH) {
    // Use shared memory to store tiles from Md and Nd
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // Calculate thread row and column within the grid
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;
    float Pvalue = 0.0f; // Accumulator for the resulting element

    // Loop over all the tiles needed for the computation
    for (int m = 0; m < WIDTH / TILE_WIDTH; m++) {
        // Load Md and Nd tiles into shared memory
        Mds[threadIdx.y][threadIdx.x] = Md[row * WIDTH + (m * TILE_WIDTH + threadIdx.x)];
        Nds[threadIdx.y][threadIdx.x] = Nd[(m * TILE_WIDTH + threadIdx.y) * WIDTH + col];
        __syncthreads(); // Ensure all threads have loaded their tile elements

        // Matrix multiplication for the current tile
        for (int k = 0; k < TILE_WIDTH; k++) {
            Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
        }
        __syncthreads(); // Ensure all threads have completed using the tiles
    }

    // Write the computed value to the output matrix
    Pd[row * WIDTH + col] = Pvalue;
}