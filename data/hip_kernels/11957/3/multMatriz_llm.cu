#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multMatriz(float *da, float *db, float *dc, int num) {
    // Calculate global row (i) and column (j) index
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    
    if (i < num && j < num) { // Ensure only valid threads perform computation
        float sum = 0; // Reset sum for each matrix element calculation

        // Perform matrix multiplication for element (i, j)
        for (unsigned int k = 0; k < num; k++) {
            sum += da[i * num + k] * db[k * num + j];
        }

        // Store result in the output matrix
        dc[i * num + j] = sum;
    }
}