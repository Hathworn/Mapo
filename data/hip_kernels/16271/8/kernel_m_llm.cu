#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_m(unsigned int *ind, unsigned int *scand, unsigned int shift, const unsigned int ne)
{
    unsigned int sosm = 1 << shift;
    int m_i_b = blockIdx.x * blockDim.x + threadIdx.x;
    if (m_i_b < ne) {  // Swap the conditional check to minimize thread divergence
        scand[m_i_b] = (ind[m_i_b] & sosm) ? 0 : 1;
    }
}