#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize using warp shuffling for performance improvement
__global__ void kernel_s(unsigned int * ind, const size_t nbn, const unsigned int ne)
{
    int m_i_b = threadIdx.x + blockIdx.x * blockDim.x;
    if (m_i_b >= ne) return;  // Bound check

    extern __shared__ float dats[];
    dats[threadIdx.x] = (m_i_b < ne) ? ind[m_i_b] : 0;
    __syncthreads();

    // Use warp shuffling to optimize the reduction process
    for (int offset = 1; offset < blockDim.x; offset *= 2) {
        float temp = 0;
        if (threadIdx.x >= offset)
            temp = dats[threadIdx.x - offset];
        __syncthreads();
        dats[threadIdx.x] += temp;
        __syncthreads();
    }

    if (m_i_b < ne) {
        if (threadIdx.x == 0) ind[blockIdx.x * blockDim.x] = 0;
        else ind[m_i_b] = dats[threadIdx.x - 1];
    }
}