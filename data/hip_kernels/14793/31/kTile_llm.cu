#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate the global index of the thread
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the total number of threads
    const int numThreads = blockDim.x * gridDim.x;

    // Loop over target elements with stride equal to the total number of threads
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Compute the y and x indices for the target
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        // Calculate the y and x indices for the source based on the modulo operation
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Assign the source value to the target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}