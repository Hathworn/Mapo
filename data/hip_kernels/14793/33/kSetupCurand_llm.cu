#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Utilize shared memory and bank conflict reduction if needed
    __shared__ hiprandState localState[NUM_RND_THREADS_PER_BLOCK];

    // Initialize local state with different sequence numbers
    hiprand_init(seed, tidx, 0, &localState[threadIdx.x]);

    // Store local state to global state
    state[tidx] = localState[threadIdx.x];
}