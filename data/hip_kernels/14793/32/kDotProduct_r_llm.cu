#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numCols, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;

    // Efficient iteration with strided accesses
    for (uint i = eidx; i < numElements; i += DP_BLOCKSIZE * gridDim.x) {
        shmem[threadIdx.x] += a[i] * b[i];
    }
    __syncthreads();

    // Warp-level reduction using efficient loop unrolling
    if (threadIdx.x < 32) {
        volatile float* mysh = &shmem[threadIdx.x];
        if (DP_BLOCKSIZE >= 64) *mysh += mysh[32];
        if (DP_BLOCKSIZE >= 32) *mysh += mysh[16];
        if (DP_BLOCKSIZE >= 16) *mysh += mysh[8];
        if (DP_BLOCKSIZE >= 8) *mysh += mysh[4];
        if (DP_BLOCKSIZE >= 4) *mysh += mysh[2];
        if (DP_BLOCKSIZE >= 2) *mysh += mysh[1];
    }
    // Store and synchronize only for the first thread in the block
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}