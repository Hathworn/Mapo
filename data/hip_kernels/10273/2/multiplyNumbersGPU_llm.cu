#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiplyNumbersGPU(float *pDataA, float *pDataB, float *pResult)
{
    int tid = (blockIdx.y * gridDim.x * blockDim.x) + blockIdx.x * blockDim.x + threadIdx.x; // Generalize grid indexing
    float a = pDataA[tid]; // Load the memory once
    float b = pDataB[tid];
    float temp = a * b / 12.34567; // Simplify arithmetic operation
    pResult[tid] = sqrt(temp) * sin(a); // Reuse already loaded variable
}