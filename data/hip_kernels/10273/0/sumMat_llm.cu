#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMat(double *A, double *B, double *C, int N)
{
    // Use shared memory to optimize memory access
    __shared__ double Asub[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double Bsub[BLOCK_SIZE][BLOCK_SIZE];

    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (col < N && row < N) {
        // Load data into shared memory for faster access
        Asub[threadIdx.y][threadIdx.x] = A[row * N + col];
        Bsub[threadIdx.y][threadIdx.x] = B[row * N + col];
        __syncthreads(); // Synchronize to ensure shared memory is loaded

        // Compute the sum
        C[row * N + col] = Asub[threadIdx.y][threadIdx.x] + Bsub[threadIdx.y][threadIdx.x];
    }
}
```
