#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Using register storage for local computation
    float local_accuracy = *accuracy; 
    local_accuracy /= N;
    *accuracy = local_accuracy;
}