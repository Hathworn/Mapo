#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate the unique thread index
    const uint tidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a unique seed for each thread
    hiprand_init(seed, tidx, 0, &state[tidx]);
}