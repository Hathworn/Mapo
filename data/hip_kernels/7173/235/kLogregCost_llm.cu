#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = __float2int_rd(labels[tx]); // Use fast intrinsic function for type conversion
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];
        
        labelLogProbs[tx] = __logf(labelp); // Lightweight log function

        // Check for equality with a small epsilon to handle floating point precision issues
        if (fabs(labelp - maxp) > 1e-6f) {
            correctProbs[tx] = 0.0f;
        } else {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                // Use atomic add to safely increment numMax
                numMax += fabs(probs[i * numCases + tx] - maxp) < 1e-6f;
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}