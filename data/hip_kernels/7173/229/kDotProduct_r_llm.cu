#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Loop unrolling for performance improvement
    for (uint i = eidx; i < numElements; i += gridDim.x * DP_BLOCKSIZE) {
        sum += a[i] * b[i];
    }

    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Reduction using warp shuffles for further optimization
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        for (int offset = 16; offset > 0; offset /= 2) {
            mysh[threadIdx.x] += mysh[threadIdx.x + offset];
        }
        if (threadIdx.x == 0) {
            target[blockIdx.x] = mysh[0];
        }
    }
}