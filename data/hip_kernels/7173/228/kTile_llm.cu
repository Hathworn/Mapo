#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate the global index once and reuse it
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    // Process elements that this thread is responsible for
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Use integer division and modulo operations to map 
        // target coordinates to source coordinates
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Transfer data from source to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}