#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Use shared memory where possible if needed for optimizations
    __shared__ int64_t shared_batch_index;
    __shared__ int64_t shared_class_index;
    if (threadIdx.x == 0) {
        shared_batch_index = batch_index;
        shared_class_index = class_index;
    }
    __syncthreads();
    
    // Optimize loop by using less arithmetic calculations
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    while (idx < num_elements) {
        int target_idx = idx * 3;
        to_normalize[target_idx] = shared_batch_index;         // Use shared memory value
        to_normalize[target_idx + 1] = shared_class_index;     // Use shared memory value
        to_normalize[target_idx + 2] = static_cast<int64_t>(original[idx]);
        idx += stride;
    }
}