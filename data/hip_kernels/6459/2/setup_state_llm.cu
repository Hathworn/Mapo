#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Use block and thread index to initialize each state separately 
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}