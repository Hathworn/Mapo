#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const int *a, const int *b, int *c, int N)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Unrolling the loop to improve performance
    int gridStride = blockDim.x * gridDim.x;
    for (int i = tid; i < N; i += gridStride)
    {
        c[i] = a[i] + b[i];
    }
}