#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dynamicReverse(int *d, int n)
{
    extern __shared__ int s[];
    int t = threadIdx.x;
    int tr = n - t - 1;

    if (t < n) { // Ensure thread index is within bounds
        s[t] = d[t];
    }
    __syncthreads();

    if (t < n) { // Ensure thread index is within bounds
        d[t] = s[tr];
    }
}