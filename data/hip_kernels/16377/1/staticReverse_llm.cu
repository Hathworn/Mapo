#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void staticReverse(int *d, int n)
{
    __shared__ int s[64];
    int t = threadIdx.x;
    int tr = n - t - 1;

    // Load data from global to shared memory for the current thread
    if(t < n) {
        s[t] = d[t];
    }
    __syncthreads();

    // Write reversed data back to global memory for the current thread
    if(t < n) {
        d[t] = s[tr];
    }
}