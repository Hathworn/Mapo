#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexSelect_contiguous(float *tensor, float *src, long stride, float *index, long idxSize)
{
    const int VT = 4;
    const int WARP_SIZE = 32;
    const int MAX_DIM_SIZE = 65535;

    int idx = blockIdx.x * blockDim.y + threadIdx.y;
    int startIdx = threadIdx.x + blockIdx.y * VT * WARP_SIZE;

    if (idx < idxSize && startIdx < stride) { // Check bounds to avoid unnecessary loops
        const long srcIdx = ((long)index[idx] - 1) * stride;
        const long targetIdx = idx * stride;

        #pragma unroll
        for (int i = 0; i < VT; i++) {
            const int featureIdx = startIdx + i * WARP_SIZE;
            if (featureIdx < stride) {
                tensor[targetIdx + featureIdx] = src[srcIdx + featureIdx];
            }
        }
    }
}