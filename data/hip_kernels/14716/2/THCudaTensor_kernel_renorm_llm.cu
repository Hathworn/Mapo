#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_renorm(float *data, const float value, const long size, const float maxnorm)
{
    __shared__ float buffer[32];
    long tx = threadIdx.x;
    long bx = blockIdx.x;
    long step = blockDim.x;
    float *row = data + size * bx;
    
    buffer[tx] = 0;

    // Calculate norm in parallel using shared memory
    for (long i = tx; i < size; i += step)
    {
        buffer[tx] += powf(fabsf(row[i]), value);
    }

    // Parallel reduction in shared memory
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
    {
        __syncthreads();
        if (tx < stride)
        {
            buffer[tx] += buffer[tx + stride];
        }
    }
    __syncthreads();

    // Check norm and renormalize if necessary
    if (tx == 0)
    {
        float norm = powf(buffer[0], 1.0f / value);
        buffer[0] = (norm > maxnorm) ? maxnorm / (norm + 1e-7f) : 1.0f;
    }
    __syncthreads();
    
    // Apply renormalization
    float scaling_factor = buffer[0];
    for (long i = tx; i < size; i += step)
    {
        if (scaling_factor != 1.0f) 
        {
            row[i] *= scaling_factor;
        }
    }
}