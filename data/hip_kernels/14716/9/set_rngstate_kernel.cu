#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_rngstate_kernel(hiprandStateMtgp32 *state, mtgp32_kernel_params *kernel)
{
state[threadIdx.x].k = kernel;
}