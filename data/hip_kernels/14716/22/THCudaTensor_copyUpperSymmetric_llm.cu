#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_copyUpperSymmetric(float *input, int n, int len)
{
    // Calculate 1D index for optimization in accessing elements
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimize stride loop by using grid stride loop
    for (int i = idx; i < len; i += gridDim.x * blockDim.x) {
        const int r = i % n;
        const int c = i / n;
        if (r > c) {
            input[i] = input[r * n + c];
        }
    }
}