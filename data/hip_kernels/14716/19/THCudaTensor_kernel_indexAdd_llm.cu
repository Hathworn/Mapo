#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexAdd(float *res, float *src, long* res_stride, float *index, long res_nDim, int dim, long idx_size, long src_size, long size_dim) {
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    long flat_size = src_size / idx_size;

    if (thread_idx < flat_size) {
        long leftover = thread_idx;  // Initialize leftover before the loop
        for (int i = 0; i < idx_size; i++) {
            int targetIdx = 0;
            int resIdx = 0;
            for (int d = 0; d < res_nDim; d++) {
                if (d == dim) {  // Skip computation when d == dim as it does not affect leftover
                    continue;
                }
                long coeff = leftover / res_stride[d];
                leftover -= coeff * res_stride[d];
                targetIdx += coeff * res_stride[d];
                resIdx += coeff * res_stride[d];
            }
            atomicAdd(&res[resIdx + ((long)(index[i]) - 1) * res_stride[dim]], src[targetIdx + i * res_stride[dim]]);
        }
    }
}