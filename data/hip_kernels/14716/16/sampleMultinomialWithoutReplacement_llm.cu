#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;
        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithoutReplacement(hiprandStateMtgp32* state, int totalSamples, int sample, float* dest, long distributions, int categories, float* origDist, float* normDistPrefixSum) {
    for (long curDistBase = blockIdx.x * blockDim.y; curDistBase < distributions; curDistBase += gridDim.x * blockDim.y) {
        long curDist = curDistBase + threadIdx.y;

        // Ensure each warp computes only one sample value
        float r = curDist < distributions ? hiprand_uniform(&state[blockIdx.x]) : 0.0f;

        // Use warp shuffle to broadcast uniform sample to all threads
        r = __shfl_sync(0xFFFFFFFF, r, 0);
        
        if (threadIdx.x == 0 && curDist < distributions) {
            int choice = binarySearchForMultinomial(normDistPrefixSum + curDist * categories, categories, r);
            dest[curDist * totalSamples + sample] = (float) choice + 1.0f;
            origDist[curDist * categories + choice] = 0.0f;
        }
    }
}