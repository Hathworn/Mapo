#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;
        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        // No probability mass or precision problems; just return the
        // first element
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithReplacement(hiprandStateMtgp32* state, int totalSamples, float* dest, long distributions, int categories, float* normDistPrefixSum) {
    // Optimize memory access by using shared memory for prefix sums
    extern __shared__ float sharedPrefixSum[];
    
    // Use cooperative threads to load distribution into shared memory
    for (long curDist = blockIdx.x; curDist < distributions; curDist += gridDim.x) {
        for (int c = threadIdx.x; c < categories; c += blockDim.x) {
            sharedPrefixSum[c] = normDistPrefixSum[curDist * categories + c];
        }
        __syncthreads();

        for (int sampleBase = 0; sampleBase < totalSamples; sampleBase += blockDim.y) {
            // The warp determines the sample
            int sample = sampleBase + threadIdx.y;

            // All threads participate in this
            float r = hiprand_uniform(&state[blockIdx.x]);

            if (threadIdx.x == 0 && sample < totalSamples) {
                // Find the bucket that a uniform sample lies in
                int choice = binarySearchForMultinomial(sharedPrefixSum, categories, r);

                // Torch indices are 1-based
                dest[curDist * totalSamples + sample] = (float) choice + 1.0f;
            }
        }
        __syncthreads();
    }
}