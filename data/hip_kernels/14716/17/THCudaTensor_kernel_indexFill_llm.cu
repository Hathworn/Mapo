#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexFill( float *tensor, long* stride, float *index, long src_nDim, int dim, long idx_size, long tensor_size, long size_dim, float val )
{
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    long flat_size = tensor_size / idx_size;
    
    if (thread_idx < flat_size)
    {
        // Optimize coefficient calculation by reducing redundant computations
        long coeff[20];  // Assume a maximum of 20 dimensions, update if needed
        long leftover = thread_idx;
        int srcIdx = 0;

        for (int d = 0; d < src_nDim; d++)
        {
            if (d < dim)
            {
                coeff[d] = leftover / (stride[d] / size_dim);
                leftover -= coeff[d] * (stride[d] / size_dim);
            }
            else if (d > dim)
            {
                coeff[d] = leftover / stride[d];
                leftover -= coeff[d] * stride[d];
            }
            else
            {
                coeff[d] = 0; // No need for calculation at dim index
            }
            srcIdx += coeff[d] * stride[d];
        }
        
        // Move index calculation outside of loop over i, as it remains constant
        long stride_dim_val = stride[dim];
        for (int i = 0; i < idx_size; i++)
        {
            tensor[srcIdx + (long)((index[i])-1) * stride_dim_val] = val;
        }
    }
}