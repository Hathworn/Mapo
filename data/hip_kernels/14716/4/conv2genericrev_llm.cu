#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv2genericrev(float *input, float *kernel, float *output, int input_n, int input_h, int input_w, int kernel_n, int kernel_h, int kernel_w, float alpha, int stride_h, int stride_w)
{
    // output dimensions
    int output_h = input_h - (kernel_h - 1) * stride_h;
    int output_w = input_w - (kernel_w - 1) * stride_w;

    // global thread indices
    int kk = blockIdx.x;
    int ii = blockIdx.y;
    int batch = threadIdx.z;
    int kid = threadIdx.x;
    int nkids = blockDim.x;
    int tid = kid + batch * nkids;
    int nthreads = blockDim.x * blockDim.z;

    // pointer offset for output
    output = output + (kk * input_n + ii) * output_h * output_w;

    // shared memory allocation
    extern __shared__ float shared_output[];

    // pointer for each thread's output in shared memory
    float *output_s = shared_output + tid * output_w * output_h;

    // convolution loop
    for (int yy = threadIdx.y; yy < output_h; yy += blockDim.y) {
        float *output_p = output_s + yy * output_w;
        for (int xx = 0; xx < output_w; xx++) {
            float *input_p = input + (ii + batch * input_n) * input_h * input_w + yy * stride_h * input_w + xx * stride_w;
            float *kernel_p = kernel + (kk + batch * kernel_n) * kernel_w * kernel_h;
            float sum = 0;
            for (int ky = 0; ky < kernel_h; ky++) {
                for (int kx = kid; kx < kernel_w; kx += nkids) {
                    // accumulate convolution sum
                    sum += input_p[kx] * kernel_p[kx];
                }
                input_p += input_w;
                kernel_p += kernel_w;
            }
            *output_p++ = sum;
        }
    }
    __syncthreads();

    // reduction across threads
    if (threadIdx.y == 0) {
        for (int k = 1; k < nthreads; k++) {
            for (int i = tid; i < output_w * output_h; i += nthreads) {
                shared_output[i] += shared_output[k * output_h * output_w + i];
            }
        }
        __syncthreads();

        // update output with scaled shared output
        for (int i = tid; i < output_w * output_h; i += nthreads) {
            output[i] += alpha * shared_output[i];
        }
    }
}