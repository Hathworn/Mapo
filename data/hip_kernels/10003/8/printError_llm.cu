#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#pragma comment(lib,"cublas.lib")

using namespace std;

//==============================Function Prototypes================================
double getRand();

__global__ void printError(float *output, float *target) {
    // Calculate global thread index
    int n = blockIdx.x * blockDim.x + threadIdx.x;

    // Remove unnecessary computation for non-existent index
    if (n < blockDim.x * gridDim.x) {
        float error = target[n] - output[n];
        printf("%f \n", error);
    }
}