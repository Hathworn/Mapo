#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#pragma comment(lib,"cublas.lib")

using namespace std;

//==============================Function Prototypes================================
double getRand();

__global__ void deltaCalcHidden(float *Activation, float *delta) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if 'n' is within valid range
    if (n < gridDim.x * blockDim.x) {
        // Store the commonly used values in local variables
        float act_val = Activation[n];
        float exp_value = exp(-act_val);
        float sigmoid = 1 / (1 + exp_value);

        // Update delta using the stored values
        delta[n] *= sigmoid * (1 - sigmoid);
    }
}