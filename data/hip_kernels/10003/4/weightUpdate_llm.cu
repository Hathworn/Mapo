#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#pragma comment(lib,"cublas.lib")

using namespace std;

//==============================Function Prototypes================================
double getRand();

__global__ void weightUpdate(float *d_W, float *d_D, float *d_N) {
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Row index
    int k = blockIdx.y * blockDim.y + threadIdx.y; // Column index
    int n = j * gridDim.y * blockDim.y + k;         // Linear index calculation

    // Check index bounds to prevent access violations
    if (n < gridDim.x * gridDim.y * blockDim.x * blockDim.y) {
        float N = 0.1; // Learning rate constant
        d_W[n] = d_W[n] + N * d_D[k] * d_N[j];
    }
}