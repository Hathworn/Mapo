#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#pragma comment(lib,"cublas.lib")

using namespace std;

//==============================Function Prototypes================================
double getRand();

__global__ void deltaCalcOutput(float *OutActivation, float *Outputdelta, float *targets) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n < blockDim.x * gridDim.x) {  // Ensure thread index is within bounds
        float activation = OutActivation[n];
        float expVal = exp(-activation);  // Compute exp(-OutActivation[n]) once
        float sigmoid = 1 / (1 + expVal); // Compute sigmoid only once
        Outputdelta[n] = (targets[n] - activation) * sigmoid * (1 - sigmoid);
    }
}