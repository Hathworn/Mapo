#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#pragma comment(lib,"cublas.lib")

using namespace std;

//==============================Function Prototypes================================
double getRand();

__global__ void initWeights(float *dst, unsigned int seed) {
    // Calculate global thread index
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure n is within bounds before processing
    if (n < seed) {
        // Optimize division and loop conditions
        dst[n] /= (float)(seed);

        // Use a more efficient approach to handle scaling condition
        if (dst[n] > 5) {
            dst[n] /= 2;
        }

        // Optimize branch handling with bitwise operations
        if ((n & (seed % 3)) == 0) {
            dst[n] = -dst[n];
        }
    }
}