#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcul_min(unsigned long *ord, int ind_start, int ind_end, unsigned long long *ymin, int *ind_min, int size_max_parallel) {

    int a = threadIdx.x;
    int size_tot = (ind_end - ind_start - 1);

    // Adjusting the number of threads and size parallel computation
    int nb_threads = (size_tot + size_max_parallel - 1) / size_max_parallel;
    int size_parallel = (size_tot + nb_threads - 1) / nb_threads; 

    int ind_start_loc = ind_start + a * size_parallel + 1;

    if (a == (nb_threads - 1)) 
        size_parallel = size_tot - a * size_parallel;

    unsigned long min_loc = ord[ind_start_loc];
    int ind_min_loc = ind_start_loc;

    // Finding local minimum in the assigned subarray
    for (int i = ind_start_loc; i < ind_start_loc + size_parallel; i++) {
        if (ord[i] < min_loc) { 
            min_loc = ord[i];
            ind_min_loc = i;
        }
    }

    // Atomic operation to find the global minimum across all threads
    atomicMin(ymin, min_loc); 

    __syncthreads();

    // Atomically update the global index of the minimum value
    if (*ymin == min_loc) {
        atomicExch(ind_min, ind_min_loc);
    }

    return;
}