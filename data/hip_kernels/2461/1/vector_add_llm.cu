#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void vector_add(float *out, float *a, float *b, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x; // Calculate linear index
    if (idx < n) { // Ensure we don't access out-of-bound elements
        out[idx] = a[idx] + b[idx]; // Perform vector addition
    }
}