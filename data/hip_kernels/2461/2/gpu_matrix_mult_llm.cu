#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_matrix_mult(float *A, float *B, float *C, int n)
{
    // Calculate row col only once
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0f; // Use local accumulator
        for (int i = 0; i < n; ++i) {
            sum += A[row * n + i] * B[i * n + col];
        }
        C[row * n + col] = sum; // Only write to C once
    }
}