#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Program Parameters */
#define N 8000  /* Matrix size */
int blocks_per_grid = 32;
int threads_per_block = 256;

/* Matrices */
float A[N*N], B[N*N];

/* CUDA arrays */
float *A_d, *B_d;

/* Optimize matrixNorm kernel function */
__global__ void matrixNorm(float* A_dd, float* B_dd, int N_d) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory to cache columns to reduce global memory access
    extern __shared__ float shared_col[];

    if (idx < N_d) {
        // Load entire column into shared memory
        for (int row = threadIdx.x; row < N_d; row += blockDim.x) {
            shared_col[row] = A_dd[row * N_d + idx];
        }
        __syncthreads();

        // Compute the mean
        float mu = 0.0;
        for (int row = 0; row < N_d; row++) {
            mu += shared_col[row];
        }
        mu /= N_d;

        // Compute the standard deviation
        float sigma = 0.0;
        for (int row = 0; row < N_d; row++) {
            sigma += powf(shared_col[row] - mu, 2.0);
        }
        sigma /= N_d;
        sigma = sqrt(sigma);

        // Normalize the column
        for (int row = threadIdx.x; row < N_d; row += blockDim.x) {
            B_dd[row * N_d + idx] = (sigma == 0.0) ? 0.0 : ((shared_col[row] - mu) / sigma);
        }
    }
}