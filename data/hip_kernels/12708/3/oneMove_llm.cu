#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define INF 2147483647

extern "C" {

}

__global__ void oneMove(int * tab, int dist, int pow, int blocksPerTask, int period) {
    __shared__ int tmp_T[1024];
    __shared__ int begin;

    // Calculate begin index once per block
    if(threadIdx.x == 0)
        begin = (blockIdx.x / blocksPerTask) * dist * 2 + (blockIdx.x % blocksPerTask) * 512 * pow;

    __syncthreads();

    int offset = (blockIdx.x / period) % 2 == 0 ? 1 : -1; // Determine sorting direction
    
    for(int i = begin; i < begin + pow * 512; i += 512) {
        int threadIdxMod = threadIdx.x % 512;
        
        // Load elements to shared memory
        if(threadIdxMod < 512) 
            tmp_T[threadIdx.x] = tab[i + threadIdx.x];
        else 
            tmp_T[threadIdx.x] = tab[i + threadIdx.x - 512 + dist];

        __syncthreads();

        // Compare and swap in shared memory
        if(threadIdxMod < 512 && (offset * tmp_T[threadIdx.x]) > (offset * tmp_T[threadIdx.x + 512])) {
            int temp = tmp_T[threadIdx.x];
            tmp_T[threadIdx.x] = tmp_T[threadIdx.x + 512];
            tmp_T[threadIdx.x + 512] = temp;
        }

        __syncthreads();

        // Store back the sorted results
        if(threadIdxMod < 512) 
            tab[i + threadIdx.x] = tmp_T[threadIdx.x];
        else 
            tab[i + threadIdx.x - 512 + dist] = tmp_T[threadIdx.x];

        __syncthreads();
    }
}