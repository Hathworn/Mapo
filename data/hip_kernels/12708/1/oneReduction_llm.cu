#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define INF 2147483647

extern "C" {

}

__global__ void oneReduction(int* tab, int len, int mod) {
    __shared__ int begin, end;
    __shared__ int tmp_T[1024];

    if (threadIdx.x == 0) {
        begin = blockIdx.x * len;
        end = begin + len;
    }

    __syncthreads();

    bool compareGreater = (blockIdx.x % mod) < (mod / 2);

    int stride = len / 2;
    while (stride >= 1024) {
        int step = 2 * stride;
        for (int g = begin; g < end; g += step) {
            for (int j = g; j < g + stride; j += 512) {
                __syncthreads();

                int localIdx = threadIdx.x;
                int offset = localIdx < 512 ? 0 : -512 + stride;
                tmp_T[localIdx] = tab[j + localIdx + offset];

                __syncthreads();

                if (localIdx < 512) {
                    int pairIdx = localIdx + 512;
                    if ((compareGreater && tmp_T[localIdx] > tmp_T[pairIdx]) ||
                        (!compareGreater && tmp_T[localIdx] < tmp_T[pairIdx])) {
                        int tmp = tmp_T[localIdx];
                        tmp_T[localIdx] = tmp_T[pairIdx];
                        tmp_T[pairIdx] = tmp;
                    }
                }

                __syncthreads();

                tab[j + localIdx + offset] = tmp_T[localIdx];
            }
        }
        stride /= 2;
    }

    for (int i = begin; i < end; i += 1024) {
        __syncthreads();
        tmp_T[threadIdx.x] = tab[i + threadIdx.x];
        __syncthreads();

        for (int jump = 512; jump >= 1; jump /= 2) {
            int pairIdx = threadIdx.x + jump;
            if (threadIdx.x < 512 && pairIdx < 1024) {
                if ((compareGreater && tmp_T[threadIdx.x] > tmp_T[pairIdx]) ||
                    (!compareGreater && tmp_T[threadIdx.x] < tmp_T[pairIdx])) {
                    int temp = tmp_T[threadIdx.x];
                    tmp_T[threadIdx.x] = tmp_T[pairIdx];
                    tmp_T[pairIdx] = temp;
                }
            }
            __syncthreads();
        }

        tab[i + threadIdx.x] = tmp_T[threadIdx.x];
    }
}