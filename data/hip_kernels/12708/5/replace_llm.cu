#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define getPos(a,k) (((a)>>(k-1))&1)

extern "C" {

}

__global__ void replace(int * input_T, int * output_T, int * prefix_T, int * prefix_helper_T, int n, int k, int blockPower) {
    int baseIndex = threadIdx.x + 1024 * blockIdx.x;  // Compute base index for each block
    int totalThreads = 1024 * gridDim.x;  // Calculate total threads
    for(int i = 0; i < blockPower; i++) {
        int oldpos = baseIndex + i * totalThreads;  // Compute old position using baseIndex
        if(oldpos >= n) return;

        int newpos = prefix_T[oldpos] + prefix_helper_T[blockIdx.x + i * gridDim.x];

        if(getPos(input_T[oldpos], k) == 0) {
            newpos = oldpos - newpos;
        } else {
            newpos = prefix_helper_T[(n + 1023) / 1024] + newpos - 1;
        }

        output_T[newpos] = input_T[oldpos];
    }
}