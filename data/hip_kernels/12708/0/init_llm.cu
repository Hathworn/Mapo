#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define INF 2147483647

extern "C" {

}

__global__ void init(int *tab, int len) {
    // Calculate the global thread index
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    // Use stride to allow each thread to process multiple elements
    int stride = blockDim.x * gridDim.x;
    for (; i < len; i += stride) {
        tab[i] = INF;
    }
}