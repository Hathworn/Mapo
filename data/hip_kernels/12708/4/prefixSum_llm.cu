#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define getPos(a,k) (((a)>>(k-1))&1)

extern "C" {

}

__global__ void prefixSum(int * input_T, int * prefix_T, int * prefix_helper_T, int n, int k, int blockPower) {
    __shared__ int tmp_T[1024];
    
    int globalIdx = threadIdx.x + 1024 * blockIdx.x;
    int gridSize = 1024 * gridDim.x;

    for(int i = 0; i < blockPower; i++) {
        int idx = globalIdx + i * gridSize;
        if (idx >= n) return;
        
        // Load input to shared memory and transform
        tmp_T[threadIdx.x] = getPos(input_T[idx], k);

        // Perform a parallel scan (Hillis-Steele)
        int val, kk = 1;
        while(kk <= 512) {
            __syncthreads();
            if (kk <= threadIdx.x) val = tmp_T[threadIdx.x - kk];
            __syncthreads();
            // Condition checked again to avoid unnecessary writes
            if (kk <= threadIdx.x) tmp_T[threadIdx.x] += val;
            kk *= 2;
        }
        
        __syncthreads();

        prefix_T[idx] = tmp_T[threadIdx.x];
        
        // Store the last element of each segment
        if (threadIdx.x == 1023 || idx == n-1) {
            prefix_helper_T[i * gridDim.x + blockIdx.x + 1] = tmp_T[threadIdx.x];
        }
    }
}