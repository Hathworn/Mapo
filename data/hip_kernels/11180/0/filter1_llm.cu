#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//-----------------------------------------
// Autor: Farias
// Data : January 2012
// Goal : Image treatment
//-----------------------------------------

/***************************************************************************************************
Includes
***************************************************************************************************/



/***************************************************************************************************
Defines
***************************************************************************************************/

#define ELEM(i,j,DIMX_) (i+(j)*(DIMX_))
#define BLOCK_SIZE 16


/***************************************************************************************************
Functions
***************************************************************************************************/

using namespace std;


/**************************************************************************************************/

__global__ void filter1( int width, int height, unsigned char *src, unsigned char *dest ) {

    // Calculate i and j using block and thread indices
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if(i > 0 && j > 0 && i < width - 1 && j < height - 1) {
        int idx_center = 3 * ELEM(i, j, width);
        int idx_left = 3 * ELEM(i - 1, j, width);
        int idx_right = 3 * ELEM(i + 1, j, width);
        int idx_top = 3 * ELEM(i, j - 1, width);
        int idx_bottom = 3 * ELEM(i, j + 1, width);
        
        for (int k = 0; k < 3; ++k) {
            int aux = 0;
            
            // Central pixel
            aux += 4 * src[idx_center + k];
            
            // Surrounding pixels
            aux += 2 * src[idx_left + k];
            aux += 2 * src[idx_right + k];
            aux += 2 * src[idx_top + k];
            aux += 2 * src[idx_bottom + k];

            aux /= 12;
            
            // Store the result
            dest[idx_center + k] = (unsigned char) aux;
        }
    }
}