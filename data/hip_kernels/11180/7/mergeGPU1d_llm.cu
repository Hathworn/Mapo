#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeGPU1d(unsigned char *image1, unsigned char *image2, unsigned char *res, int pixels) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < pixels) {
        int idx = 3 * i;

        // Perform the average calculation with integer summation to avoid type casting
        res[idx + 2] = (unsigned char)((image1[idx + 2] + image2[idx + 2]) >> 1);
        res[idx + 1] = (unsigned char)((image1[idx + 1] + image2[idx + 1]) >> 1);
        res[idx    ] = (unsigned char)((image1[idx    ] + image2[idx    ]) >> 1);
    }
}