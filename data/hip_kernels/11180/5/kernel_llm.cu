#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel( int *a, int *b, int *c ) {

    // Calculate thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure idx is within bounds
    if (idx < N) {

        // Pre-calculate boundary indices
        int idx1 = idx + 1;
        int idx2 = idx + 2;

        // Use shared memory to reduce global memory accesses
        __shared__ int shared_a[256];
        __shared__ int shared_b[256];

        shared_a[threadIdx.x] = a[idx];
        shared_b[threadIdx.x] = b[idx];

        // Ensure boundary conditions
        if (threadIdx.x == blockDim.x - 1) {
            shared_a[(threadIdx.x + 1) % 256] = a[idx1 % N];
            shared_b[(threadIdx.x + 1) % 256] = b[idx1 % N];
            shared_a[(threadIdx.x + 2) % 256] = a[idx2 % N];
            shared_b[(threadIdx.x + 2) % 256] = b[idx2 % N];
        }

        __syncthreads();

        // Compute average using preloaded shared memory
        float as = (shared_a[threadIdx.x] + shared_a[(threadIdx.x + 1) % 256] + shared_a[(threadIdx.x + 2) % 256]) / 3.0f;
        float bs = (shared_b[threadIdx.x] + shared_b[(threadIdx.x + 1) % 256] + shared_b[(threadIdx.x + 2) % 256]) / 3.0f;
        
        // Store result in global memory
        c[idx] = (as + bs) / 2;
    }
}