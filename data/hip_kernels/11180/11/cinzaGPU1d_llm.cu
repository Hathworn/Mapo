#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cinzaGPU1d(unsigned char *image1, unsigned char *res, int pixels) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < pixels) {
        int idx = 3 * i;
        // Load color values once into registers
        int r = image1[idx + 2];
        int g = image1[idx + 1];
        int b = image1[idx];
        
        // Compute gray scale value using integer arithmetic
        int cinza = (30 * r + 59 * g + 11 * b) / 100;
        unsigned char gray = static_cast<unsigned char>(cinza);

        // Set output values
        res[idx] = gray;
        res[idx + 1] = gray;
        res[idx + 2] = gray;
    }
}