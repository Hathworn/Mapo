#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use loop unrolling to process multiple elements per thread
    int step = blockDim.x * gridDim.x;
    
    for(; i < numElements; i += step)
    {
        C[i] = A[i] + B[i];
    }
}