#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeGPU1d( unsigned char *image1, unsigned char *image2, unsigned char *res, int pixels ) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // Use built-in vectorized operations for better performance and readability
    if (i < pixels) {
        int idx = 3 * i;
        int4 pixel1 = make_int4(image1[idx], image1[idx + 1], image1[idx + 2], 0);
        int4 pixel2 = make_int4(image2[idx], image2[idx + 1], image2[idx + 2], 0);

        // Calculate the average directly using int operations
        int4 avg = make_int4((pixel1.z + pixel2.z) >> 1,
                             (pixel1.y + pixel2.y) >> 1,
                             (pixel1.x + pixel2.x) >> 1, 
                             0);

        res[idx] = static_cast<unsigned char>(avg.x);
        res[idx + 1] = static_cast<unsigned char>(avg.y);
        res[idx + 2] = static_cast<unsigned char>(avg.z);
    }
}