#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/***************************************************************************************************
Defines
***************************************************************************************************/

#define ELEM(i,j,DIMX_) (i+(j)*(DIMX_))
#define BLOCK_SIZE 16

/***************************************************************************************************
Functions
***************************************************************************************************/

__global__ void filter2( int width, int height, unsigned char *src, unsigned char *dest ) {

    // Calculate global thread coordinates
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    __shared__ int pesos[3][3];

    // Initialize shared weights matrix once
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        pesos[0][0] = 0; pesos[0][1] = 2; pesos[0][2] = 0;
        pesos[1][0] = 2; pesos[1][1] = 4; pesos[1][2] = 2;
        pesos[2][0] = 0; pesos[2][1] = 2; pesos[2][2] = 0;
    }

    __syncthreads(); // Ensure all threads have the initialized weights

    if (i > 0 && j > 0 && i < width - 1 && j < height - 1) {
        for (int k = 0; k < 3; ++k) {
            int aux = 0;
            for (int lin = 0; lin < 3; lin++) {
                for (int col = 0; col < 3; col++) {
                    // Access image data and accumulate result
                    int idx = 3 * ELEM(i + lin - 1, j + col - 1, width);
                    aux += pesos[lin][col] * src[idx + k];
                }
            }
            aux /= 12; // Normalize the filter result
            // Store result in destination array
            int idx = 3 * ELEM(i, j, width);
            dest[idx + k] = (unsigned char)aux;
        }
    }
}