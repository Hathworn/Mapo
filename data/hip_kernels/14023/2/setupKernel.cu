#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setupKernel(hiprandState *state, unsigned long long seed) {
int idx = threadIdx.x + blockDim.x * blockIdx.x;
hiprand_init(seed, idx, 0, &state[idx]);
}