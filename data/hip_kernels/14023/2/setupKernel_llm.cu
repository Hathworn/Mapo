#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to initialize the RNG state
__global__ void setupKernel(hiprandState *state, unsigned long long seed) {
    // Compute the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize random number generator state
    hiprand_init(seed, idx, 0, &state[idx]);
}