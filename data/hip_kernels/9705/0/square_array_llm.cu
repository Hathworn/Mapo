#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for grid-stride loop
    for (int i = idx; i < N; i += stride) // Use grid-stride loop to increase parallelism
    {
        a[i] = a[i] * a[i];
    }
}