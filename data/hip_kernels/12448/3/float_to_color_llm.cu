#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void float_to_color(uchar4 * pixels, float* in) {
    // Calculate global thread index
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // Load input value
    float num = in[offset];

    // Pre-calculate terms used multiple times
    int scaledNum = static_cast<int>(num * 255);
    int scaledMaxTemp = static_cast<int>((MAX_TEMP - num) * 255);

    // Assign to pixel
    pixels[offset].x = scaledNum;
    pixels[offset].y = 0;
    pixels[offset].z = scaledMaxTemp;
    pixels[offset].w = 255;
}