#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CalculateFixed(const float *subBG, const float *subT, const int *subM, float *fixed, const int wb, const int hb, const int wt, const int ht, const int oy, const int ox) {
    // Precompute directional and coefficient values
    __shared__ int dir[16][2];
    __shared__ int coef[16];
    if (threadIdx.x < 16) {
        dir[threadIdx.x][0] = (threadIdx.x < 3) ? threadIdx.x * 2 - 2 : (threadIdx.x - 4) % 3 - 1;
        dir[threadIdx.x][1] = (threadIdx.x < 3) ? -2 : ((threadIdx.x < 6) ? -1 : ((threadIdx.x == 8 || threadIdx.x == 12) ? 0 : ((threadIdx.x < 15) ? 1 : 2)));
        coef[threadIdx.x] = (threadIdx.x % 3 == 0) ? 1 : 2;
    }
    __syncthreads();

    const int num = 24;
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    const int curt = wt * yt + xt;

    if (yt < ht && xt < wt) {
        float sum[3] = {0}, bsum[3] = {0};

        // Loop through all directions
        for (int i = 0; i < 16; i++) {
            int dxt = xt + dir[i][0];
            int dyt = yt + dir[i][1];
            int dcurt = wt * dyt + dxt;
            int dxb = ox + dxt;
            int dyb = oy + dyt;

            bool inBounds = (dxt >= 0 && dxt < wt && dyt >= 0 && dyt < ht);

            // Texture background contributions
            float coef_val = coef[i];
            sum[0] += subT[(inBounds ? dcurt : curt) * 3 + 0] * coef_val;
            sum[1] += subT[(inBounds ? dcurt : curt) * 3 + 1] * coef_val;
            sum[2] += subT[(inBounds ? dcurt : curt) * 3 + 2] * coef_val;

            // Background contributions
            if (!inBounds || !subM[inBounds ? dcurt : curt]) {
                dxb = max(min(dxb, wb - 1), 0);
                dyb = max(min(dyb, hb - 1), 0);
                int dcurb = wb * dyb + dxb;
                bsum[0] += subBG[dcurb * 3 + 0] * coef_val;
                bsum[1] += subBG[dcurb * 3 + 1] * coef_val;
                bsum[2] += subBG[dcurb * 3 + 2] * coef_val;
            }
        }

        // Compute fixed values
        fixed[curt * 3 + 0] = subT[curt * 3 + 0] - sum[0] / num + bsum[0] / num;
        fixed[curt * 3 + 1] = subT[curt * 3 + 1] - sum[1] / num + bsum[1] / num;
        fixed[curt * 3 + 2] = subT[curt * 3 + 2] - sum[2] / num + bsum[2] / num;
    }
}