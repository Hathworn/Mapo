#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CalculateSampleB(const float *background, float *subBG, const int wb, const int hb, const int ws, const int hs, const int sRate) {
    // Calculate current sample position
    const int ys = blockIdx.y * blockDim.y + threadIdx.y;
    const int xs = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within the sample space
    if (ys < hs && xs < ws) {
        const int curst = ws * ys + xs;
        const int yb = ys * sRate;
        const int xb = xs * sRate;
        
        int num = 0;
        float sum[3] = {0.0f, 0.0f, 0.0f};

        // Improved memory coalescing by adjusting index calculation
        for (int i = 0; i < sRate; ++i) {
            for (int j = 0; j < sRate; ++j) {
                int y_offset = yb + i;
                int x_offset = xb + j;
                if (y_offset < hb && x_offset < wb) {
                    int curb = wb * y_offset + x_offset;
                    sum[0] += background[curb * 3 + 0];
                    sum[1] += background[curb * 3 + 1];
                    sum[2] += background[curb * 3 + 2];
                    ++num;
                }
            }
        }
        
        // Assign averaged color values
        subBG[curst * 3 + 0] = sum[0] / num;
        subBG[curst * 3 + 1] = sum[1] / num;
        subBG[curst * 3 + 2] = sum[2] / num;
    }
}