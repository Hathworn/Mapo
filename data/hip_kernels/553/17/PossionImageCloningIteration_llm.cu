#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PossionImageCloningIteration(const float *fixed, const int *mask, float *input, float *output, const int wt, const int ht) {
    const int dir[16][2] = {{-2, -2},    {0, -2},    {2, -2},
                            {-1, -1},    {0, -1},    {1, -1},
                            {-2, 0},     {-1, 0},    {1, 0},     {2, 0},
                            {-1, 1},     {0, 1},     {1, 1},
                            {-2, 2},     {0, 2},     {2, 2}};
    const int coef[16] = {1, 1, 1,
                          2, 2, 2,
                          1, 2, 2, 1,
                          2, 2, 2,
                          1, 1, 1};
    const float inv_num = 1.0f / 24.0f;  // Precompute inverse of num

    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    const int curt = wt * yt + xt;

    if (yt < ht && xt < wt) {
        float sum[3] = {0.0f, 0.0f, 0.0f};
        
        #pragma unroll  // Unroll the loop to increase efficiency
        for (int i = 0; i < 16; i++) {
            int dxt = xt + dir[i][0];
            int dyt = yt + dir[i][1];
            if (dxt >= 0 && dxt < wt && dyt >= 0 && dyt < ht) {
                int dcurt = wt * dyt + dxt;
                if (mask[dcurt]) {
                    float coefVal = static_cast<float>(coef[i]);
                    sum[0] += input[dcurt * 3 + 0] * coefVal;
                    sum[1] += input[dcurt * 3 + 1] * coefVal;
                    sum[2] += input[dcurt * 3 + 2] * coefVal;
                }
            }
        }

        output[curt * 3 + 0] = fixed[curt * 3 + 0] + sum[0] * inv_num;
        output[curt * 3 + 1] = fixed[curt * 3 + 1] + sum[1] * inv_num;
        output[curt * 3 + 2] = fixed[curt * 3 + 2] + sum[2] * inv_num;
    }
}