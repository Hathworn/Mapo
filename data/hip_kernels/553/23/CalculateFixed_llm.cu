#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CalculateFixed(const float *background, const float *target, const float *mask, float *fixed, const int wb, const int hb, const int wt, const int ht, const int oy, const int ox) {
    const int dir[4][2] = { {0, -1}, {1, 0}, {0, 1}, {-1, 0}};
    int num = 4;
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    const int curt = wt * yt + xt;
    
    if (yt < ht && xt < wt && mask[curt] > 127.0f) {
        float sum[3] = {0.0f, 0.0f, 0.0f};
        float bsum[3] = {0.0f, 0.0f, 0.0f};
        
        // Reduce recalculations of indices
        int target_base = curt * 3;
        float target_val[3] = { target[target_base], target[target_base + 1], target[target_base + 2] };
        
        for (int i = 0; i < 4; i++) {
            int dxt = xt + dir[i][0], dyt = yt + dir[i][1];
            
            if (dxt >= 0 && dxt < wt && dyt >= 0 && dyt < ht) {
                int dcurt = wt * dyt + dxt;
                int dtarget_base = dcurt * 3;
                
                sum[0] += target[dtarget_base];
                sum[1] += target[dtarget_base + 1];
                sum[2] += target[dtarget_base + 2];
                
                if (mask[dcurt] >= 127.0f) continue;
            } else {
                sum[0] += target_val[0];
                sum[1] += target_val[1];
                sum[2] += target_val[2];
            }
            
            int dxb = max(min(ox + dxt, wb - 1), 0);
            int dyb = max(min(oy + dyt, hb - 1), 0);
            int dcurb = wb * dyb + dxb;
            int dbackground_base = dcurb * 3;
            
            bsum[0] += background[dbackground_base];
            bsum[1] += background[dbackground_base + 1];
            bsum[2] += background[dbackground_base + 2];
        }

        fixed[target_base] = target_val[0] - sum[0] / num + bsum[0] / num;
        fixed[target_base + 1] = target_val[1] - sum[1] / num + bsum[1] / num;
        fixed[target_base + 2] = target_val[2] - sum[2] / num + bsum[2] / num;
    }
}