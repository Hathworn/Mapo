#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CalculateSampleT(const float *target, const float *mask, float *subT, int *subM, const int wt, const int ht, const int ws, const int hs, const int sRate) {
    // Calculate global indices
    const int ys = blockIdx.y * blockDim.y + threadIdx.y;
    const int xs = blockIdx.x * blockDim.x + threadIdx.x;
    const int curst = ws * ys + xs;

    if (ys < hs && xs < ws) {
        const int yt = ys * sRate;
        const int xt = xs * sRate;
        int num = 0;
        float sum[3] = {0.0f, 0.0f, 0.0f};
        int _or = 0;

        // Unroll loops for better performance
        #pragma unroll
        for (int i = 0; i < sRate; i++) {
            #pragma unroll
            for (int j = 0; j < sRate; j++) {
                if (yt + i < ht && xt + j < wt) {
                    int curt = wt * (yt + i) + (xt + j);
                    sum[0] += target[curt * 3 + 0];
                    sum[1] += target[curt * 3 + 1];
                    sum[2] += target[curt * 3 + 2];
                    _or |= (mask[curt] > 127.0f);
                    num++;
                }
            }
        }

        subM[curst] = _or;
        subT[curst * 3 + 0] = sum[0] / num; // Use assignment instead of accumulation
        subT[curst * 3 + 1] = sum[1] / num;
        subT[curst * 3 + 2] = sum[2] / num;
    }
}