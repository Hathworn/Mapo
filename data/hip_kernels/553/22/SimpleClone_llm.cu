#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SimpleClone(const float *background, const float *target, const float *mask, float *output, const int wb, const int hb, const int wt, const int ht, const int oy, const int ox) 
{
    // Calculate thread location in grid
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Avoid unnecessary computations
    if (yt >= ht || xt >= wt) return;
    
    const int curt = wt * yt + xt;
    if (mask[curt] > 127.0f) {
        const int yb = oy + yt, xb = ox + xt;
        
        // Check if inside valid boundary once
        if (yb >= 0 && yb < hb && xb >= 0 && xb < wb) {
            const int curb = wb * yb + xb;
            // Coalesce memory accesses for output
            for (int i = 0; i < 3; ++i) {
                output[curb * 3 + i] = target[curt * 3 + i];
            }
        }
    }
}