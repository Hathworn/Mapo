#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalculateSampleB(const float *background, float *subBG, const int wb, const int hb, const int ws, const int hs, const int sRate) {
    const int ys = blockIdx.y * blockDim.y + threadIdx.y;
    const int xs = blockIdx.x * blockDim.x + threadIdx.x;
    const int curst = ws * ys + xs;

    if (ys < hs && xs < ws) {
        const int yb = ys * sRate;
        const int xb = xs * sRate;
        int num = 0;
        float sum[3] = {0};

        // Use registers to reduce global memory accesses
        #pragma unroll
        for (int i = 0; i < sRate; i++) {
            #pragma unroll
            for (int j = 0; j < sRate; j++) {
                if (yb + i < hb && xb + j < wb) {
                    int curb = wb * (yb + i) + (xb + j);
                    float3 pixel = make_float3(background[curb * 3 + 0], background[curb * 3 + 1], background[curb * 3 + 2]);
                    sum[0] += pixel.x;
                    sum[1] += pixel.y;
                    sum[2] += pixel.z;
                    num++;
                }
            }
        }
        float inv_num = 1.0f / num; // Precompute inverse for division
        subBG[curst * 3 + 0] = sum[0] * inv_num;
        subBG[curst * 3 + 1] = sum[1] * inv_num;
        subBG[curst * 3 + 2] = sum[2] * inv_num;
    }
}