#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SimpleClone(const float *background, const float *target, const int *mask, float *output, const int wb, const int hb, const int wt, const int ht, const int oy, const int ox)
{
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    if (yt < ht && xt < wt) {
        const int curt = wt * yt + xt;
        if (mask[curt]) {
            const int yb = oy + yt, xb = ox + xt;
            // Merge condition checks to reduce branching
            if (yb >= 0 && yb < hb && xb >= 0 && xb < wb) {
                const int curb = wb * yb + xb;
                const int targetIdx = curt * 3;
                const int outputIdx = curb * 3;
                // Coalesced memory access
                output[outputIdx + 0] = target[targetIdx + 0];
                output[outputIdx + 1] = target[targetIdx + 1];
                output[outputIdx + 2] = target[targetIdx + 2];
            }
        }
    }
}