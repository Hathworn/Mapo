#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CalculateTransSample(const float *input, float *output, const int wtss, const int htss, const int wts, const int hts, const int ratio) {
    const int yts = blockIdx.y * blockDim.y + threadIdx.y;
    const int xts = blockIdx.x * blockDim.x + threadIdx.x;
    const int curst = wts * yts + xts;
    const int yt = yts * ratio;
    const int xt = xts * ratio;

    if (yts < hts && xts < wts) {
        // Loop unrolling for better performance
        int idx;
        for (int i = 0; i < ratio; i++) {
            idx = yt + i;
            if (idx < htss) {
                for (int j = 0; j < ratio; j++) {
                    int jdx = xt + j;
                    if (jdx < wtss) {
                        const int curt = wtss * idx + jdx;
                        output[curt * 3 + 0] = input[curst * 3 + 0];
                        output[curt * 3 + 1] = input[curst * 3 + 1];
                        output[curt * 3 + 2] = input[curst * 3 + 2];
                    }
                }
            }
        }
    }
}