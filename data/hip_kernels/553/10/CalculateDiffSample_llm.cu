#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CalculateDiffSample(float *cur, float *pre, const int wts, const int hts) {
    // Use shared memory to reduce global memory access latency
    extern __shared__ float shared_pre[];

    const int yts = blockIdx.y * blockDim.y + threadIdx.y;
    const int xts = blockIdx.x * blockDim.x + threadIdx.x;
    const int curst = wts * yts + xts;

    if (yts < hts && xts < wts) {
        // Load pre data into shared memory
        int sharedIdx = threadIdx.y * blockDim.x + threadIdx.x;
        shared_pre[sharedIdx*3+0] = pre[curst*3+0];
        shared_pre[sharedIdx*3+1] = pre[curst*3+1];
        shared_pre[sharedIdx*3+2] = pre[curst*3+2];
        __syncthreads();

        // Calculate difference using shared memory
        cur[curst*3+0] -= shared_pre[sharedIdx*3+0];
        cur[curst*3+1] -= shared_pre[sharedIdx*3+1];
        cur[curst*3+2] -= shared_pre[sharedIdx*3+2];

        // Reset pre array element values
        pre[curst*3+0] = 0;
        pre[curst*3+1] = 0;
        pre[curst*3+2] = 0;
    }
}