#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SimpleClone(const float *background, const float *target, const int *mask, float *output, const int wb, const int hb, const int wt, const int ht, const int oy, const int ox)
{
    // Calculate thread position
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if out of bounds
    if (yt >= ht || xt >= wt) return;

    const int curt = wt * yt + xt;

    // Proceed if the mask is non-zero
    if (mask[curt]) {
        const int yb = oy + yt, xb = ox + xt;
        const int curb = wb * yb + xb;

        // Only proceed with memory operations if indices are in bounds
        if (yb >= 0 && yb < hb && xb >= 0 && xb < wb) {
            // Vectorize the output assignments
            float3 temp = *((float3*)&target[curt * 3]);
            *((float3*)&output[curb * 3]) = temp;
        }
    }
}