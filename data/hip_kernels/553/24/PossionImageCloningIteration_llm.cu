#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PossionImageCloningIteration(const float *fixed, const float *mask, float *input, float *output, const int wt, const int ht) {
    const int dir[4][2] = {{0, -1}, {1, 0}, {0, 1}, {-1, 0}};
    int num = 4;
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    const int curt = wt * yt + xt;

    // Optimization: Early return if thread is out of bounds or mask value is insufficient
    if (yt >= ht || xt >= wt || mask[curt] <= 127.0f) return;

    float sum[3] = {0};

    // Unrolling loop for performance improvement
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        int dxt = xt + dir[i][0];
        int dyt = yt + dir[i][1];
        int dcurt = wt * dyt + dxt;
        if (dxt >= 0 && dxt < wt && dyt >= 0 && dyt < ht && mask[dcurt] > 127.0f) {
            sum[0] += input[dcurt * 3 + 0];
            sum[1] += input[dcurt * 3 + 1];
            sum[2] += input[dcurt * 3 + 2];
        }
    }

    // Pre-compute index to reduce repetitive calculations
    int outputIndex = curt * 3;
    output[outputIndex + 0] = fixed[outputIndex + 0] + sum[0] / num;
    output[outputIndex + 1] = fixed[outputIndex + 1] + sum[1] / num;
    output[outputIndex + 2] = fixed[outputIndex + 2] + sum[2] / num;
}