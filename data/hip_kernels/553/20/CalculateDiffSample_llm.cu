#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalculateDiffSample( float *cur, float *pre, const int wts, const int hts ){
    // Calculate global thread index
    const int yts = blockIdx.y * blockDim.y + threadIdx.y;
    const int xts = blockIdx.x * blockDim.x + threadIdx.x;
    const int curst = wts * yts + xts;

    // Use if condition to ensure threads within bounds execute
    if (yts < hts && xts < wts) {
        // Use registers to reduce redundant memory access
        float pre_val_0 = pre[curst*3+0];
        float pre_val_1 = pre[curst*3+1];
        float pre_val_2 = pre[curst*3+2];

        cur[curst*3+0] -= pre_val_0;
        cur[curst*3+1] -= pre_val_1;
        cur[curst*3+2] -= pre_val_2;

        // Zero out 'pre' values directly
        pre[curst*3+0] = 0.0f;
        pre[curst*3+1] = 0.0f;
        pre[curst*3+2] = 0.0f;
    }
}