#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalculateTransSample( const float *input, float *output, const int wtss, const int htss, const int wts, const int hts, const int ratio ) {
    const int yts = blockIdx.y * blockDim.y + threadIdx.y;
    const int xts = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit if thread is out of bounds
    if (yts >= hts || xts >= wts) return;

    const int curst = wts * yts + xts;
    const int yt = yts * ratio;
    const int xt = xts * ratio;

    // Unroll the loop if ratio is small and known at compile time (e.g., ratio == 2 or 4)
    #pragma unroll
    for (int i = 0; i < ratio; i++) {
        #pragma unroll
        for (int j = 0; j < ratio; j++) {
            int yt_i = yt + i;
            int xt_j = xt + j;
            if (yt_i < htss && xt_j < wtss) {
                const int curt = wtss * yt_i + xt_j;
                output[curt * 3 + 0] = input[curst * 3 + 0];
                output[curt * 3 + 1] = input[curst * 3 + 1];
                output[curt * 3 + 2] = input[curst * 3 + 2];
            }
        }
    }
}