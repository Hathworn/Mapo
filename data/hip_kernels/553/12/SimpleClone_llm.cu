#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SimpleClone(const float *background, const float *target, const float *mask, float *output, const int wb, const int hb, const int wt, const int ht, const int oy, const int ox)
{
    // Calculate target indices
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    if (yt >= ht || xt >= wt) return;  // Exit early if out of bounds

    const int curt = wt * yt + xt;

    // Early exit if mask condition is not met
    if (mask[curt] <= 127.0f) return;

    // Calculate background indices
    const int yb = oy + yt, xb = ox + xt;
    if (yb < 0 || yb >= hb || xb < 0 || xb >= wb) return;  // Exit early if out of bounds

    const int curb = wb * yb + xb;

    // Directly copy target values to output
    output[curb * 3 + 0] = target[curt * 3 + 0];
    output[curb * 3 + 1] = target[curt * 3 + 1];
    output[curb * 3 + 2] = target[curt * 3 + 2];
}