#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CalculateFixed(const float* subBG, const float* subT, const int* subM, float* fixed, const int wb, const int hb, const int wt, const int ht, const int oy, const int ox) {
    const int dir[16][2] = {
        {-2, -2}, {0, -2}, {2, -2},
        {-1, -1}, {0, -1}, {1, -1},
        {-2, 0}, {-1, 0}, {1, 0}, {2, 0},
        {-1, 1}, {0, 1}, {1, 1},
        {-2, 2}, {0, 2}, {2, 2}
    };
    const int coef[16] = {1, 1, 1, 2, 2, 2, 1, 2, 2, 1, 2, 2, 2, 1, 1, 1};
    const int num = 24;
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    const int curt = wt * yt + xt;

    if (yt < ht && xt < wt) {
        float sum[3] = {0.0f};
        float bsum[3] = {0.0f};

        for (int i = 0; i < 16; ++i) {
            int dxt = xt + dir[i][0];
            int dyt = yt + dir[i][1];
            int dcurt = wt * dyt + dxt;

            int dxb = ox + dxt;
            int dyb = oy + dyt;

            bool withinBounds = (dxt >= 0 && dxt < wt && dyt >= 0 && dyt < ht);

            // Conditional accumulation for subT based on bounds
            sum[0] += ((withinBounds) ? subT[dcurt * 3 + 0] : subT[curt * 3 + 0]) * coef[i];
            sum[1] += ((withinBounds) ? subT[dcurt * 3 + 1] : subT[curt * 3 + 1]) * coef[i];
            sum[2] += ((withinBounds) ? subT[dcurt * 3 + 2] : subT[curt * 3 + 2]) * coef[i];

            if (!withinBounds || !subM[dcurt]) {
                // Clamp boundaries for subBG index
                dxb = max(0, min(dxb, wb - 1));
                dyb = max(0, min(dyb, hb - 1));
                int dcurb = wb * dyb + dxb;

                // Accumulate weighted subBG values
                bsum[0] += subBG[dcurb * 3 + 0] * coef[i];
                bsum[1] += subBG[dcurb * 3 + 1] * coef[i];
                bsum[2] += subBG[dcurb * 3 + 2] * coef[i];
            }
        }

        // Compute fixed values normalized by num
        fixed[curt * 3 + 0] = subT[curt * 3 + 0] - sum[0] / num + bsum[0] / num;
        fixed[curt * 3 + 1] = subT[curt * 3 + 1] - sum[1] / num + bsum[1] / num;
        fixed[curt * 3 + 2] = subT[curt * 3 + 2] - sum[2] / num + bsum[2] / num;
    }
}