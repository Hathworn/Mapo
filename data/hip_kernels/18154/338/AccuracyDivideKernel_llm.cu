#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to compute the accuracy
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread 0 to perform the division to prevent race conditions
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
    // Optionally store and synchronize if necessary, to ensure the operation is done correctly
    __syncthreads();
}