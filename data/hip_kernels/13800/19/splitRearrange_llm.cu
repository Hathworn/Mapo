#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void splitRearrange (int numElems, int iter, unsigned int* keys_i, unsigned int* keys_o, unsigned int* values_i, unsigned int* values_o, unsigned int* histo) {
    // Use shared memory for histogram and temporary arrays
    __shared__ unsigned int histo_s[(1<<BITS)];
    __shared__ unsigned int array_s[4*SORT_BS];

    int index = blockIdx.x*4*SORT_BS + 4*threadIdx.x;

    // Load histogram data to shared memory
    if (threadIdx.x < (1<<BITS)){
        histo_s[threadIdx.x] = histo[gridDim.x*threadIdx.x+blockIdx.x];
    }
    
    // Efficient conditional loading of data
    uint4 mine = {UINT32_MAX, UINT32_MAX, UINT32_MAX, UINT32_MAX};
    uint4 value;
    if (index < numElems) {
        mine = *((uint4*)(keys_i+index));
        value = *((uint4*)(values_i+index));
    }

    // Calculate masks
    uint4 masks = {
        (mine.x&((1<<(BITS*(iter+1)))-1))>>(BITS*iter),
        (mine.y&((1<<(BITS*(iter+1)))-1))>>(BITS*iter),
        (mine.z&((1<<(BITS*(iter+1)))-1))>>(BITS*iter),
        (mine.w&((1<<(BITS*(iter+1)))-1))>>(BITS*iter)
    };
    
    ((uint4*)array_s)[threadIdx.x] = masks;
    __syncthreads();

    // Use shared memory for histogram computations
    uint4 new_index = {histo_s[masks.x], histo_s[masks.y], histo_s[masks.z], histo_s[masks.w]};

    // Unroll loop for improving performance
    int i = 4*threadIdx.x - 1;
    if (i >= 0) {
        if (array_s[i] == masks.x) new_index.x++;
        i--;
    }
    if (i >= 0) {
        if (array_s[i] == masks.x) new_index.x++;
    }

    // Sequence the index increments
    new_index.y = (masks.y == masks.x) ? new_index.x+1 : new_index.y;
    new_index.z = (masks.z == masks.y) ? new_index.y+1 : new_index.z;
    new_index.w = (masks.w == masks.z) ? new_index.z+1 : new_index.w;

    // Efficiently write results back to global memory
    if (index < numElems){
        keys_o[new_index.x] = mine.x;
        values_o[new_index.x] = value.x;

        keys_o[new_index.y] = mine.y;
        values_o[new_index.y] = value.y;

        keys_o[new_index.z] = mine.z;
        values_o[new_index.z] = value.z;

        keys_o[new_index.w] = mine.w;
        values_o[new_index.w] = value.w;
    }
}