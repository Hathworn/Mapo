#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mysgemmNT(const float *A, int lda, const float *B, int ldb, float *C, int ldc, int k, float alpha, float beta)
{
    // Use shared memory to optimize memory access patterns for A and B
    __shared__ float Asub[32][32]; 
    __shared__ float Bsub[32][32]; 

    float c = 0.0f;
    int m = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y;

    for (int i = 0; i < k; i += 32) {
        // Load data into shared memory tile
        Asub[threadIdx.y][threadIdx.x] = A[m + (i + threadIdx.y) * lda];
        Bsub[threadIdx.y][threadIdx.x] = B[n + (i + threadIdx.x) * ldb];
        __syncthreads();
 
        // Use shared memory to compute tiles
        for (int j = 0; j < 32; ++j) {
            c += Asub[threadIdx.y][j] * Bsub[j][threadIdx.x];
        }
        __syncthreads();
    }

    // Write the final result to the global memory C
    C[m + n * ldc] = C[m + n * ldc] * beta + alpha * c;
}