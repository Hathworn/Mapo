#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define ITER 4
#define BANK_OFFSET1(n) (n) + (((n) >> 5))
#define BANK_OFFSET(n) (n) + (((n) >> 5))
#define NUM_BLOCKS(length, dim) nextPow2(length) / (2 * dim)
#define ELEM 4
#define TOTAL_THREADS 512
#define TWO_PWR(n) (1 << (n))
extern float toBW(int bytes, float sec);

__device__ __inline__ void prefix_sum_warp(int thid, int* temp, int N)
{
    if (thid < 16)
    {
        int i = temp[thid];
        if (thid >= 1) temp[thid] += temp[thid - 1];
        if (thid >= 2) temp[thid] += temp[thid - 2];
        if (thid >= 4) temp[thid] += temp[thid - 4];
        if (thid >= 8) temp[thid] += temp[thid - 8];
        temp[thid] -= i;
    }
}

__global__ void output_index(
    int* device_hist, int* pdevice_hist, 
    int* device_input, int* device_out, 
    int length, int num_blocks, int nibble)
{
    __shared__ int temp[TWO_PWR(ITER)];

    int t = 4 * blockIdx.x * blockDim.x + threadIdx.x;
    int thid = threadIdx.x;

    if (t < length)
    {
        // Pre-fetch data into registers to minimize memory access latency
        int val1 = device_input[t];
        int val2 = device_input[t + TOTAL_THREADS];
        int val3 = device_input[t + 2 * TOTAL_THREADS];
        int val4 = device_input[t + 3 * TOTAL_THREADS];
        
        // Load histogram data into shared memory, unrolling loop to minimize overhead
        if (thid < 32 && thid < ITER) 
        {
            temp[4 * thid] = device_hist[4 * thid * num_blocks + blockIdx.x];
            temp[4 * thid + 1] = device_hist[(4 * thid + 1) * num_blocks + blockIdx.x];
            temp[4 * thid + 2] = device_hist[(4 * thid + 2) * num_blocks + blockIdx.x];
            temp[4 * thid + 3] = device_hist[(4 * thid + 3) * num_blocks + blockIdx.x];
        }

        prefix_sum_warp(thid, temp, TWO_PWR(ITER));
        __syncthreads();

        // Calculate indices using pre-computed shared memory values
        int nibble1 = nibble << 2;
        int mask = (1 << ITER) - 1;
        int lindex1 = temp[(val1 >> nibble1) & mask];
        int lindex2 = temp[(val2 >> nibble1) & mask];
        int lindex3 = temp[(val3 >> nibble1) & mask];
        int lindex4 = temp[(val4 >> nibble1) & mask];
        int gindex1 = pdevice_hist[((val1 >> nibble1) & mask) * num_blocks + blockIdx.x];
        int gindex2 = pdevice_hist[((val2 >> nibble1) & mask) * num_blocks + blockIdx.x];
        int gindex3 = pdevice_hist[((val3 >> nibble1) & mask) * num_blocks + blockIdx.x];
        int gindex4 = pdevice_hist[((val4 >> nibble1) & mask) * num_blocks + blockIdx.x];

        // Write output values using calculated global indices
        device_out[thid + gindex1 - lindex1] = val1;
        device_out[thid + TOTAL_THREADS + gindex2 - lindex2] = val2;
        device_out[thid + 2 * TOTAL_THREADS + gindex3 - lindex3] = val3;
        device_out[thid + 3 * TOTAL_THREADS + gindex4 - lindex4] = val4;
    }
}