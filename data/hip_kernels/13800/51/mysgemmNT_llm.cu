#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mysgemmNT(const float *A, int lda, const float *B, int ldb, float* C, int ldc, int k, float alpha, float beta) {
    // Partial results
    float c[TILE_N] = {0.0f};
    int mid = threadIdx.y * blockDim.x + threadIdx.x; // Flattened id
    int m = blockIdx.x * TILE_M + mid;
    int n = blockIdx.y * TILE_N + threadIdx.x;
    
    // Use shared memory to store tile of B
    __shared__ float b_s[TILE_TB_HEIGHT][TILE_N];

    for (int i = 0; i < k; i += TILE_TB_HEIGHT) {
        // Load the tile of B into shared memory
        if (n < TILE_N) { 
            b_s[threadIdx.y][threadIdx.x] = B[n + (i+threadIdx.y)*ldb];
        }
        __syncthreads();

        // Compute partial results
        for (int j = 0; j < TILE_TB_HEIGHT; j++) {
            float a = (m < TILE_M) ? A[m + (i+j)*lda] : 0.0f; // Avoid out of bounds
            for (int kk = 0; kk < TILE_N; kk++) {
                c[kk] += a * b_s[j][kk];
            }
        }
        __syncthreads();
    }

    // Write results back with bounds checks
    if (m < TILE_M) {
        int t = ldc * blockIdx.y * TILE_N + m;
        for (int i = 0; i < TILE_N; i++) {
            if ((t+i*ldc) < ldc) { // Ensure within C matrix bounds
                C[t+i*ldc] = C[t+i*ldc] * beta + alpha * c[i];
            }
        }
    }
}