#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputePhiMag_GPU(float* phiR, float* phiI, float* phiMag, int numK) {
    int indexK = blockIdx.x * blockDim.x + threadIdx.x; // Utilize blockDim.x for more flexibility
    if (indexK < numK) {
        // Use local variables to avoid redundant memory accesses
        float real = phiR[indexK];
        float imag = phiI[indexK];
        phiMag[indexK] = real * real + imag * imag;
    }
}