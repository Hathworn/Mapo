#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uniformAdd(unsigned int n, unsigned int *data, unsigned int *inter) 
{
    // Utilize shared memory to efficiently load the uniform value
    __shared__ unsigned int uni;
    if (threadIdx.x == 0) { 
        uni = inter[blockIdx.x]; 
    }
    __syncthreads();

    // Calculate the global index using threadIdx, blockIdx, and blockDim
    unsigned int g_ai = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // Perform addition if the global index is within the valid range
    if (g_ai < n) {
        data[g_ai] += uni;  // Handle the first element
        // Check and handle the second element only if it's within bounds
        if (g_ai + blockDim.x < n) {  
            data[g_ai + blockDim.x] += uni;
        }
    }
}