#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define ITER 4
#define BANK_OFFSET1(n) (n) + (((n) >> 5))
#define BANK_OFFSET(n) (n) + (((n) >> 5))
#define NUM_BLOCKS(length, dim) nextPow2(length) / (2 * dim)
#define ELEM 4
#define TOTAL_THREADS 512
#define TWO_PWR(n) (1 << (n))
extern float toBW(int bytes, float sec);

__global__ void add_kernel(int* device_result, int* device_blocksum)
{
    int thid = threadIdx.x;
    int N = blockDim.x;
    int offset = blockIdx.x * 4 * blockDim.x;
    int temp1 = device_blocksum[blockIdx.x];

    // Use loop unrolling to reduce loop overhead
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        device_result[offset + thid + i * N] += temp1;
    }
}