#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel_2elements(int* device_result, int* device_blocksum_2elements)
{
    __shared__ int temp1;
    int thid = threadIdx.x;
    int block_offset = blockIdx.x * blockDim.x * 4; // Calculate block offset

    if (thid == 0) temp1 = device_blocksum_2elements[blockIdx.x];
    __syncthreads();

    // Reduce memory calculations by using a single variable for offset
    int base_index = block_offset + thid;
    int N = blockDim.x;

    for (int i = 0; i < 4; i++) {
        device_result[base_index + i * N] += temp1; // Optimize repeated calculations
    }
}