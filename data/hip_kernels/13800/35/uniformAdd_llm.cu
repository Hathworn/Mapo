#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(unsigned int n, unsigned int *data, unsigned int *inter)
{
    __shared__ unsigned int uni;
    
    // Load shared data only by the first thread
    if (threadIdx.x == 0) {
        uni = inter[blockIdx.x];
    }
    
    __syncthreads();

    // Use grid-stride loop to enable more efficient memory access and handle different sizes
    for (unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x; 
         i < n; 
         i += blockDim.x * gridDim.x * 2) {
        if (i < n) data[i] += uni;
        if (i + blockDim.x < n) data[i + blockDim.x] += uni;
    }
}