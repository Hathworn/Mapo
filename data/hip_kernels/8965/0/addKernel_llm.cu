#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel
__global__ void addKernel(int *c, const int *a) {
    int i = threadIdx.x;
    extern __shared__ int seme[];

    // Load shared memory and synchronize
    seme[i] = a[i];
    __syncthreads();

    // Use warp-level reduction for efficiency
    int lane = i % warpSize;

    // Task based on thread index
    switch(i) {
        case 0:
            c[0] = 0;
            for (int d = lane; d < 5; d += warpSize) {
                c[0] += seme[d] * seme[d];
            }
            break;
        case 1:
            c[1] = 0;
            for (int d = lane; d < 5; d += warpSize) {
                c[1] += seme[d];
            }
            break;
        case 2:
            c[2] = 1;
            for (int d = lane; d < 5; d += warpSize) {
                c[2] *= seme[d];
            }
            break;
    }
}