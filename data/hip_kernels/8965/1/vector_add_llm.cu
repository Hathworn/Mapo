#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vector_add(int *a, int *b, int *c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we do not access out-of-bounds memory
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}