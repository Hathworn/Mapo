#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel with multiple threads initialization
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Calculate unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Initialize state per thread
    hiprand_init(seed, idx, 0, &state[idx]);
}