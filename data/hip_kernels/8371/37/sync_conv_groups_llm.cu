#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function for better performance and readability
__global__ void sync_conv_groups() {
    // Sync threads to ensure all warps in the block are ready
    __syncthreads();
}