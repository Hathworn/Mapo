#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void remove_redness_from_coordinates(const unsigned int* d_coordinates, unsigned char* d_r, unsigned char* d_b, unsigned char* d_g, unsigned char* d_r_output, int num_coordinates, int num_pixels_y, int num_pixels_x, int template_half_height, int template_half_width)
{
    int ny = num_pixels_y;
    int nx = num_pixels_x;
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;
    int imgSize = nx * ny;

    // Check if global thread index is within the number of coordinate limits
    if (global_index_1d < num_coordinates)
    {
        unsigned int image_index_1d = d_coordinates[imgSize - global_index_1d - 1];
        ushort2 image_index_2d = make_ushort2(image_index_1d % nx, image_index_1d / nx);

        // Unroll loop to reduce loop overhead 
        for (int y_offset = -template_half_height; y_offset <= template_half_height; y_offset++)
        {
            for (int x_offset = -template_half_width; x_offset <= template_half_width; x_offset++)
            {
                int x = image_index_2d.x + x_offset;
                int y = image_index_2d.y + y_offset;

                // Condense the clamping logic using min/max with ternary
                x = (x < 0) ? 0 : ((x >= nx) ? (nx - 1) : x);
                y = (y < 0) ? 0 : ((y >= ny) ? (ny - 1) : y);

                int image_offset_index_1d_clamped = y * nx + x;

                unsigned char g_value = d_g[image_offset_index_1d_clamped];
                unsigned char b_value = d_b[image_offset_index_1d_clamped];

                d_r_output[image_offset_index_1d_clamped] = (unsigned char)((g_value + b_value) / 2);
            }
        }
    }
}