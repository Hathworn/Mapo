#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
{
    // Calculate normalization constant
    const float normalization_constant = 1.f / d_input_cdf[n - 1];

    // Calculate global thread index
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    // Process elements within bounds
    if (global_index_1d < n)
    {
        // Fetch input, normalize, and store result
        d_output_cdf[global_index_1d] = d_input_cdf[global_index_1d] * normalization_constant;
    }
}