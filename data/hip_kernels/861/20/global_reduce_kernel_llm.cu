#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
    __shared__ float sharedData[1024]; // Use shared memory for reduction
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // Load data into shared memory and synchronize
    sharedData[tid] = d_in[myId];
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sharedData[tid] += sharedData[tid + s];
        }
        __syncthreads(); // Make sure all adds at one stage are done
    }

    // Only thread 0 writes result for this block back to global memory
    if (tid == 0)
    {
        d_out[blockIdx.x] = sharedData[0];
    }
}