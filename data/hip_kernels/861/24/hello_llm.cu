#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Store threadIdx.x in a register to reduce global memory access
    int tid = threadIdx.x;

    // Use blockIdx.y to demonstrate potential block-level parallelism
    printf("Hello world! I'm thread %d in block %d\n", tid, blockIdx.y);
}