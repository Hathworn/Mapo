#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use a single thread in each block to print
    if (threadIdx.x == 0)
    {
        printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
    }
}