#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int *s) {
    unsigned int tid = threadIdx.x;  // Get the thread index
    s[tid] = p;                      // Store the element in shared memory
    __syncthreads();                 // Synchronize to ensure all loads are complete

    // Warp-level reduction
    if (tid < 16) s[tid] += s[tid + 16];
    if (tid < 8) s[tid] += s[tid + 8];
    if (tid < 4) s[tid] += s[tid + 4];
    if (tid < 2) s[tid] += s[tid + 2];
    if (tid == 0) s[0] += s[1];

    return s[0];  // Return the result stored in s[0]
}

__global__ void reduce(unsigned int *d_out_shared, const unsigned int *d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    if (t == 0) {
        *d_out_shared = sr;
    }
}