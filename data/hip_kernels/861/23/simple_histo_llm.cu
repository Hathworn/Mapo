#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;

    // Ensure the thread index does not exceed the input size
    if (myId < BIN_COUNT) {  
        int myItem = d_in[myId];
        int myBin = myItem % BIN_COUNT;
        
        // Use atomicAdd to update bins without race conditions
        atomicAdd(&(d_bins[myBin]), 1);
    }
}