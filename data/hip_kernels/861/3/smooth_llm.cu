#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth(float * v_new, const float * v) {
    // Calculate the global index for the current thread
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimized calculation of left and right indices using modulo for boundary conditions
    int myLeftIdx = (myIdx - 1 + blockDim.x * gridDim.x) % (blockDim.x * gridDim.x);
    int myRightIdx = (myIdx + 1) % (blockDim.x * gridDim.x);

    // Retrieve elements at the computed indices
    float myElt = v[myIdx];
    float myLeftElt = v[myLeftIdx];
    float myRightElt = v[myRightIdx];

    // Calculate the new value and write it to output
    v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
}