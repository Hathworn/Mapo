#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naive_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    extern __shared__ int s_bins[]; // Use shared memory for faster access
    int tid = threadIdx.x;
    int myId = tid + blockDim.x * blockIdx.x;
    
    if (tid < BIN_COUNT) {
        s_bins[tid] = 0; // Initialize shared memory bins
    }
    __syncthreads();

    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;
    atomicAdd(&s_bins[myBin], 1); // Use atomic operations to avoid race conditions
    __syncthreads();

    if (tid < BIN_COUNT) {
        atomicAdd(&d_bins[tid], s_bins[tid]); // Write results back to global memory
    }
}