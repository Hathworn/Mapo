#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shmem_reduce_kernel(float * d_out, const float * d_in)
{
    // Using extern shared memory for reduction
    extern __shared__ float sdata[];

    // Compute global indices
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // Load from global to shared memory
    sdata[tid] = d_in[myId];
    __syncthreads(); // Ensure all loads are done

    // Unroll the last warp for better performance
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // Sync after each step
    }

    if (tid < 32) {
        // Unrolling warp: no need for __syncthreads() within a warp
        if (blockDim.x >= 64) sdata[tid] += sdata[tid + 32];
        if (blockDim.x >= 32) sdata[tid] += sdata[tid + 16];
        if (blockDim.x >= 16) sdata[tid] += sdata[tid + 8];
        if (blockDim.x >= 8)  sdata[tid] += sdata[tid + 4];
        if (blockDim.x >= 4)  sdata[tid] += sdata[tid + 2];
        if (blockDim.x >= 2)  sdata[tid] += sdata[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}