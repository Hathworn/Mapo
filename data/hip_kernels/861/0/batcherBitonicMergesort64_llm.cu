#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// http://en.wikipedia.org/wiki/Bitonic_sort

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid  = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    for (int stage = 0; stage <= 5; stage++)
    {
        int distance = 1 << stage;
        for (int substage = stage; substage >= 0; substage--)
        {
            int pairDistance = distance >> substage;
            int index = tid ^ pairDistance;
            
            // Ensure index is in bounds and that values should indeed be swapped
            if (index > tid)
            {
                // Swap elements based on direction
                if ((tid & distance) == 0)
                {
                    // Ascending order
                    if (sdata[tid] > sdata[index])
                    {
                        float temp = sdata[tid];
                        sdata[tid] = sdata[index];
                        sdata[index] = temp;
                    }
                }
                else
                {
                    // Descending order
                    if (sdata[tid] < sdata[index])
                    {
                        float temp = sdata[tid];
                        sdata[tid] = sdata[index];
                        sdata[index] = temp;
                    }
                }
            }
            __syncthreads();  // Synchronize at each substage
        }
    }

    d_out[tid] = sdata[tid];
}