#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    const int dimThread = 16;  // Number of threads working concurrently on x-axis block
    int numPts = 0;
    
    // Calculate block's starting x-coordinate and step through every 16 lines
    for (int y = bx; y < h / 32; y += gridDim.x) {
        for (int x = 0; x < w; x += dimThread) {
            unsigned int val = minmax[y * w + x + tx];
            if (val) {
                for (int k = 0; k < 32 && numPts < maxPts; k++) {
                    if (val & 0x1) {
                        // Calculate pointer position and increment safely if within maxPts
                        int ptrIdx = atomicAdd(&numPts, (numPts < maxPts) ? 1 : 0);
                        if (ptrIdx < maxPts) {
                            ptrs[dimThread * ptrIdx + tx] = (y * 32 + k) * w + x + tx;
                        }
                    }
                    val >>= 1;  // Right shift the bits of val
                }
            }
        }
    }
}