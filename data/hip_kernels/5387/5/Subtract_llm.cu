#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;  // Use blockDim instead of custom SUBTRACT_W
    const int y = blockIdx.y * blockDim.y + threadIdx.y;  // Use blockDim instead of custom SUBTRACT_H

    if (x < width && y < height) {  // Move index calculation inside the bounds check
        int p = y * pitch + x;
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}