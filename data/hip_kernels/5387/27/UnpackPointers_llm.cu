#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    const int gridStride = gridDim.x * blockDim.x;
    int numPts = 0;

    // Process grid-stride loop to allow for larger problem sizes
    for (int tid = blockIdx.x * blockDim.x + tx; tid < (h/32) * (w/16); tid += gridStride) {
        int y = tid / (w/16);
        int x = (tid % (w/16)) * 16;

        unsigned int val = minmax[y * w + x + tx];
        
        if (val) {
            int localNumPts = 0;

            #pragma unroll // Suggest the use of loop unrolling for known small iteration counts
            for (int k = 0; k < 32; k++) {
                if ((val & 0x1) && (localNumPts + numPts < maxPts)) {
                    ptrs[(numPts + localNumPts) * 16 + tx] = (y * 32 + k) * w + x + tx;
                    localNumPts++;
                }
                val >>= 1;
            }
            
            numPts += localNumPts; // Update numPts outside of the loop
        }
    }
}