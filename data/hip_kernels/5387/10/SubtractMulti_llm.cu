#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;  // Use blockDim for more flexibility
    const int y = blockIdx.y * blockDim.y + threadIdx.y;  // Use blockDim for more flexibility
    int sz = height * pitch;
    int p = threadIdx.z * sz + y * pitch + x;
    if (x < width && y < height)
    {
        d_Result[p] = d_Data[p] - d_Data[p + sz];  // Avoid branching by using logic directly
    }
    // Remove unnecessary __syncthreads() for potential performance gain
}