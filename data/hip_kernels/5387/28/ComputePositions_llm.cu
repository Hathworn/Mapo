#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputePositions(float *g_Data1, float *g_Data2, float *g_Data3, int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
    // Calculate global thread index
    int i = blockIdx.x * POSBLK_SIZE + threadIdx.x;

    // Check if the thread index is within bounds
    if (i >= numPts)
        return;

    int p = d_Ptrs[i];

    // Pre-fetch values from memory into registers to reduce latency
    float val0 = g_Data2[p];
    float val1 = g_Data2[p - 1];
    float val2 = g_Data2[p + 1];
    float val3 = g_Data2[p - w];
    float val4 = g_Data2[p + w];
    float val5 = g_Data3[p];
    float val6 = g_Data1[p];

    // Calculate first derivatives
    float dx = 0.5f * (val2 - val1);
    float dy = 0.5f * (val4 - val3);
    float ds = 0.5f * (val6 - val5);

    // Calculate second derivatives
    float dxx = 2.0f * val0 - val1 - val2;
    float dyy = 2.0f * val0 - val3 - val4;
    float dss = 2.0f * val0 - val5 - val6;

    // Calculate mixed derivatives
    float dxy = 0.25f * (g_Data2[p + w + 1] + g_Data2[p - w - 1] - g_Data2[p - w + 1] - g_Data2[p + w - 1]);
    float dxs = 0.25f * (g_Data3[p + 1] + g_Data1[p - 1] - g_Data1[p + 1] - g_Data3[p - 1]);
    float dys = 0.25f * (g_Data3[p + w] + g_Data1[p - w] - g_Data3[p - w] - g_Data1[p + w]);

    // Calculate determinant and its inverse
    float idxx = dyy * dss - dys * dys;
    float idxy = dys * dxs - dxy * dss;
    float idxs = dxy * dys - dyy * dxs;
    float idyy = dxx * dss - dxs * dxs;
    float idys = dxy * dxs - dxx * dys;
    float idss = dxx * dyy - dxy * dxy;
    float det = idxx * dxx + idxy * dxy + idxs * dxs;
    float idet = 1.0f / det;

    // Calculate position offsets
    float pdx = idet * (idxx * dx + idxy * dy + idxs * ds);
    float pdy = idet * (idxy * dx + idyy * dy + idys * ds);
    float pds = idet * (idxs * dx + idys * dy + idss * ds);

    // Boundary checks for position adjustments
    if (pdx < -0.5f || pdx > 0.5f || pdy < -0.5f || pdy > 0.5f || pds < -0.5f || pds > 0.5f) {
        pdx = __fdividef(dx, dxx);
        pdy = __fdividef(dy, dyy);
        pds = __fdividef(ds, dss);
    }

    // Compute value adjustments
    float dval = 0.5f * (dx * pdx + dy * pdy + ds * pds);

    // Store results in d_Sift
    d_Sift[i + 0 * maxPts] = (p % w) + pdx;
    d_Sift[i + 1 * maxPts] = (p / w) + pdy;
    d_Sift[i + 2 * maxPts] = d_ConstantA[0] * exp2f(pds * d_ConstantB[0]);
    d_Sift[i + 3 * maxPts] = val0 + dval;

    // Compute trace and determinant for Hessian
    float tra = dxx + dyy;
    det = dxx * dyy - dxy * dxy;
    
    // Store final Hessian ratio in d_Sift
    d_Sift[i + 4 * maxPts] = __fdividef(tra * tra, det);
}