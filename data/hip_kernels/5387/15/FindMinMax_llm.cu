#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * blockDim.x + tx; // Use blockDim.x
    const int y = blockIdx.y * blockDim.y; // Consistent usage of blockDim.y
    int p = y * pitch + x;

    // Use early return to reduce indentation
    if (x >= width) {
        return;
    }

    float val = d_Data[p];
    minvals[tx] = val;
    maxvals[tx] = val;
    
    for (int ty = 1; ty < blockDim.y; ty++) { // Use blockDim.y
        p += pitch;
        if (x < width) { // Maintain bounds check
            val = d_Data[p];
            if (val < minvals[tx]) minvals[tx] = val; // Unroll conditions
            if (val > maxvals[tx]) maxvals[tx] = val;
        }
    }
    __syncthreads();

    // Optimize reduction loop with iteration limiting
    for (int d = b / 2; d > 0; d >>= 1) { // Use bitwise shift for division
        if (tx < d) {
            float temp_min = minvals[tx + d]; // Use temporary variables
            float temp_max = maxvals[tx + d];
            if (temp_min < minvals[tx]) minvals[tx] = temp_min;
            if (temp_max > maxvals[tx]) maxvals[tx] = temp_max;
        }
        __syncthreads();
    }

    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr + 0] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}