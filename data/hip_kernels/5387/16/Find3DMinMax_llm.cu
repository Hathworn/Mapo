#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel
__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int pitch, int height) {
    __shared__ float data1[3 * (MINMAX_W + 2)];
    __shared__ float data2[3 * (MINMAX_W + 2)];
    __shared__ float data3[3 * (MINMAX_W + 2)];
    __shared__ float ymin1[(MINMAX_W + 2)];
    __shared__ float ymin2[(MINMAX_W + 2)];
    __shared__ float ymin3[(MINMAX_W + 2)];
    __shared__ float ymax1[(MINMAX_W + 2)];
    __shared__ float ymax2[(MINMAX_W + 2)];
    __shared__ float ymax3[(MINMAX_W + 2)];

    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_W;
    const int xEndClamped = min(xStart + MINMAX_W - 1, width - 1);
    const int xReadPos = xStart + tx - WARP_SIZE;
    const int xWritePos = xStart + tx;
    const int memWid = MINMAX_W + 2;

    int memPos0 = tx - WARP_SIZE + 1;
    int memPos1 = memPos0;
    int yq = 0;
    unsigned int output = 0;

    for (int y = 0; y < 32 + 2; y++) {
        output >>= 1;
        int yp = min(max(32 * blockIdx.y + y - 1, 0), height - 1);
        int readStart = yp * pitch;

        if (tx >= WARP_SIZE - 1) {
            float data_val1 = (xReadPos < 0 || xReadPos >= width) ? 0 : d_Data1[readStart + xReadPos];
            float data_val2 = (xReadPos < 0 || xReadPos >= width) ? 0 : d_Data2[readStart + xReadPos];
            float data_val3 = (xReadPos < 0 || xReadPos >= width) ? 0 : d_Data3[readStart + xReadPos];
            
            data1[yq * memWid + memPos0] = data_val1;
            data2[yq * memWid + memPos0] = data_val2;
            data3[yq * memWid + memPos0] = data_val3;
        }
        __syncthreads();

        if (y > 1 && tx < memWid) {
            float min1 = fminf(fminf(data1[memPos0], data1[memPos1]), data1[yq * memWid + tx]);
            float min2 = fminf(fminf(data2[memPos0], data2[memPos1]), data2[yq * memWid + tx]);
            float min3 = fminf(fminf(data3[memPos0], data3[memPos1]), data3[yq * memWid + tx]);
            float max1 = fmaxf(fmaxf(data1[memPos0], data1[memPos1]), data1[yq * memWid + tx]);
            float max2 = fmaxf(fmaxf(data2[memPos0], data2[memPos1]), data2[yq * memWid + tx]);
            float max3 = fmaxf(fmaxf(data3[memPos0], data3[memPos1]), data3[yq * memWid + tx]);

            ymin1[tx] = min1;
            ymin2[tx] = fminf(fminf(min1, min2), min3);
            ymin3[tx] = min3;
            ymax1[tx] = max1;
            ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
            ymax3[tx] = max3;
        }
        __syncthreads();

        if (y > 1 && tx < MINMAX_W && xWritePos <= xEndClamped) {
            float minv = fminf(
                fminf(fminf(ymin2[tx], ymin2[tx+2]), ymin1[tx+1]),
                fminf(ymin3[tx+1], fminf(data2[memPos0+1], data2[yq * memWid + tx+1]))
            );
            minv = fminf(minv, d_Threshold[1]);

            float maxv = fmaxf(
                fmaxf(fmaxf(ymax2[tx], ymax2[tx+2]), ymax1[tx+1]),
                fmaxf(ymax3[tx+1], fmaxf(data2[memPos0+1], data2[yq * memWid + tx+1]))
            );
            maxv = fmaxf(maxv, d_Threshold[0]);

            if (data2[memPos1 + 1] < minv || data2[memPos1 + 1] > maxv) {
                output |= 0x80000000;
            }
        }
        __syncthreads();

        memPos0 = memPos1;
        memPos1 = yq * memWid + tx;
        yq = (yq < 2 ? yq + 1 : 0);
    }

    if (tx < MINMAX_W && xWritePos < width) {
        int writeStart = blockIdx.y * pitch + xWritePos;
        d_Result[writeStart] = output;
    }
}