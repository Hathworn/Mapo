#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height) {
    // Use blockDim and blockIdx to calculate x, y directly to improve readability and slightly enhance performance.
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;

    // Perform conditional check and computation, without using deprecated __mul24.
    if (x < width && y < height) {
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
}