#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Calculate the global index
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;
    
    // Perform subtraction if within bounds
    if (x < width && y < height)
    {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}