#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int height)
{
    __shared__ float data1[3 * (MINMAX_SIZE + 2)];
    __shared__ float data2[3 * (MINMAX_SIZE + 2)];
    __shared__ float data3[3 * (MINMAX_SIZE + 2)];
    __shared__ float ymin1[(MINMAX_SIZE + 2)];
    __shared__ float ymin2[(MINMAX_SIZE + 2)];
    __shared__ float ymin3[(MINMAX_SIZE + 2)];
    __shared__ float ymax1[(MINMAX_SIZE + 2)];
    __shared__ float ymax2[(MINMAX_SIZE + 2)];
    __shared__ float ymax3[(MINMAX_SIZE + 2)];

    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_SIZE;
    const int xEndClamped = min(xStart + MINMAX_SIZE - 1, width - 1);
    const int xReadPos = xStart + tx - WARP_SIZE;
    const int xWritePos = xStart + tx;
    const int memWid = MINMAX_SIZE + 2;

    int memPos0 = tx - WARP_SIZE + 1;
    int memPos1 = memPos0;
    int yq = 0;
    unsigned int output = 0;
    
    for (int y = 0; y < 34; y++) {
        output >>= 1;
        int memPos = yq * memWid + memPos0;
        int yp = max(min(32 * blockIdx.y + y - 1, height - 1), 0);
        int readStart = yp * width;

        // Load data into shared memory if within bounds
        if (tx >= WARP_SIZE - 1) {
            if (xReadPos < 0 || xReadPos >= width) {
                data1[memPos] = data2[memPos] = data3[memPos] = 0;
            } else {
                data1[memPos] = d_Data1[readStart + xReadPos];
                data2[memPos] = d_Data2[readStart + xReadPos];
                data3[memPos] = d_Data3[readStart + xReadPos];
            }
        }
        __syncthreads();

        int memPos2 = yq * memWid + tx;
        if (y > 1) {
            if (tx < memWid) {
                // Min and Max calculations
                float min1 = min(data1[memPos0], min(data1[memPos1], data1[memPos2]));
                float min2 = min(data2[memPos0], min(data2[memPos1], data2[memPos2]));
                float min3 = min(data3[memPos0], min(data3[memPos1], data3[memPos2]));
                float max1 = max(data1[memPos0], max(data1[memPos1], data1[memPos2]));
                float max2 = max(data2[memPos0], max(data2[memPos1], data2[memPos2]));
                float max3 = max(data3[memPos0], max(data3[memPos1], data3[memPos2]));
                
                ymin1[tx] = min1;
                ymin2[tx] = min(min1, min(min2, min3));
                ymin3[tx] = min3;
                ymax1[tx] = max1;
                ymax2[tx] = max(max1, max(max2, max3));
                ymax3[tx] = max3;
            }
        }
        __syncthreads();

        if (y > 1) {
            if (tx < MINMAX_SIZE) {
                if (xWritePos <= xEndClamped) {
                    // Final min and max checks
                    float minv = min(min(ymin2[tx], ymin2[tx + 2]), min(ymin1[tx + 1], min(ymin3[tx + 1], min(data2[memPos0 + 1], data2[memPos2 + 1]))));
                    minv = min(minv, d_ConstantA[1]);
                    float maxv = max(max(ymax2[tx], ymax2[tx + 2]), max(ymax1[tx + 1], max(ymax3[tx + 1], max(data2[memPos0 + 1], data2[memPos2 + 1]))));
                    maxv = max(maxv, d_ConstantA[0]);
                    
                    if (data2[memPos1 + 1] < minv || data2[memPos1 + 1] > maxv) {
                        output |= 0x80000000;
                    }
                }
            }
        }
        __syncthreads();

        memPos0 = memPos1;
        memPos1 = memPos2;
        yq = (yq < 2) ? yq + 1 : 0;
    }
    
    if (tx < MINMAX_SIZE && xWritePos < width) {
        int writeStart = blockIdx.y * width + xWritePos;
        d_Result[writeStart] = output;
    }
}