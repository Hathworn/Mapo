#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOrientations(float *g_Data, float *d_Sift, int maxPts, int w, int h)
{
    // Increase tile size for better data reuse
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Initialize histogram bins
    for (int i = 0; i < 13; i++)
        hist[i * 32 + tx] = 0.0f;
    __syncthreads();
    
    // Precompute Gaussian values only once
    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    if (tx < 15)
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));
    
    // Use shared memory for frequently accessed data
    int xp = (int)(d_Sift[bx + 0 * maxPts] - 6.5f);
    int yp = (int)(d_Sift[bx + 1 * maxPts] - 6.5f);
    int px = xp & 15;
    int x = tx - px;
    
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = xp + x;
        int yi = yp + y;
        xi = max(0, min(xi, w - 1));  // Bound check in one line
        yi = max(0, min(yi, h - 1));  // Bound check in one line
        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];  // Load data into shared memory
    }
    __syncthreads();
    
    // Compute gradients and histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = 16.0f * atan2f(dy, dx) / 3.1416f + 16.5f;
            bin = bin < 32 ? bin : 0;  // Wrap bin index
            float grad = sqrtf(dx * dx + dy * dy);
            hist[32 * (x - 1) + bin] += grad * gauss[x] * gauss[y];
        }
    }
    __syncthreads();
    
    // Reduce histogram to find peaks
    for (int y = 0; y < 5; y++)
        hist[y * 32 + tx] += hist[(y + 8) * 32 + tx];
    __syncthreads();
    for (int y = 0; y < 4; y++)
        hist[y * 32 + tx] += hist[(y + 4) * 32 + tx];
    __syncthreads();
    for (int y = 0; y < 2; y++)
        hist[y * 32 + tx] += hist[(y + 2) * 32 + tx];
    __syncthreads();
    hist[tx] += hist[32 + tx];
    __syncthreads();

    // Compute smoothed histograms
    if (tx < 32) {
        if (tx == 0)
            hist[32] = 6 * hist[0] + 4 * (hist[1] + hist[31]) + (hist[2] + hist[30]);
        else if (tx == 1)
            hist[33] = 6 * hist[1] + 4 * (hist[2] + hist[0]) + (hist[3] + hist[31]);
        else if (tx <= 29)
            hist[tx + 32] = 6 * hist[tx] + 4 * (hist[tx + 1] + hist[tx - 1]) +
                (hist[tx + 2] + hist[tx - 2]);
        else if (tx == 30)
            hist[62] = 6 * hist[30] + 4 * (hist[31] + hist[29]) + (hist[0] + hist[28]);
        else if (tx == 31)
            hist[63] = 6 * hist[31] + 4 * (hist[0] + hist[30]) + (hist[1] + hist[29]);
    }
    __syncthreads();

    // Find peaks in histogram
    float v = hist[32 + tx];
    hist[tx] = (v > hist[32 + ((tx + 1) & 31)] && v >= hist[32 + ((tx + 31) & 31)] ? v : 0.0f);
    __syncthreads();

    if (tx == 0) {
        float maxval1 = 0.0;
        float maxval2 = 0.0;
        int i1 = -1;
        int i2 = -1;
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }
        float val1 = hist[32 + ((i1 + 1) & 31)];
        float val2 = hist[32 + ((i1 + 31) & 31)];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Sift[bx + 5 * maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);

        if (maxval2 < 0.8f * maxval1)
            i2 = -1;
        if (i2 >= 0) {
            float val1 = hist[32 + ((i2 + 1) & 31)];
            float val2 = hist[32 + ((i2 + 31) & 31)];
            float peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Sift[bx + 6 * maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        } else
            d_Sift[bx + 6 * maxPts] = i2;
    }
}