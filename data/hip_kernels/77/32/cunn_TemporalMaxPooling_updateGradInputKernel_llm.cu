#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Precompute offsets for efficiency
    int index_offset = blockIdx.x * input_w * input_n + (
                    threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n * dW;
    int output_offset = blockIdx.x * output_w * input_n + (
                    threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n;

    // Determine if current thread is within bounds
    int current_index = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    if (current_index < output_w) {
        // Load data pointers
        float *gradInput_data = gradInput + index_offset;
        float *gradOutput_data = gradOutput + output_offset;
        float *indices_data = indices + output_offset;

        // For all features
        for (int feat = 0; feat < input_n; ++feat) {
            // Direct indexing into arrays for efficiency
            int index = static_cast<int>(indices_data[feat]) * input_n + feat;
            gradInput_data[index] += gradOutput_data[feat];
        }
    }
}