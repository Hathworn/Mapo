#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_ClassNLLCriterion_updateGradInput_kernel(float *gradInput, float *target, int nframe, int ndim, float grad, int ntarget) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    if (index < nframe) {  // Ensure threads don't exceed the number of frames
        for (int j = 0; j < ntarget; ++j) {
            int t = static_cast<int>(target[index * ntarget + j]) - 1;
            if (t >= 0) {
                gradInput[index * ndim + t] = grad;  // Set grad input if valid
            }
        }
    }
}