#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w) {
    // Compute global thread indices for reduced loop calculations
    int xx_start = blockDim.x * blockIdx.x + threadIdx.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;

    if (xx_start < output_w && yy_start < output_h) {
        // Locate the corresponding positions in memory
        float *ptr_gradOutput = gradOutput + (blockIdx.x * output_w * output_h) + yy_start * output_w + xx_start;
        float *ptr_ind_x = indices_x + (blockIdx.x * output_w * output_h) + yy_start * output_w + xx_start;
        float *ptr_ind_y = indices_y + (blockIdx.x * output_w * output_h) + yy_start * output_w + xx_start;

        int x_start = (int)floorf(float(xx_start) / output_w * input_w);
        int y_start = (int)floorf(float(yy_start) / output_h * input_h);

        float z = *ptr_gradOutput;
        int argmax_x = (*ptr_ind_x)-1;
        int argmax_y = (*ptr_ind_y)-1;

        // Use atomicAdd to avoid race conditions
        float *ptr_gradInput = gradInput + (blockIdx.x * input_w * input_h) + (y_start + argmax_y) * input_w + (x_start + argmax_x);
        atomicAdd(ptr_gradInput, z);
    }
}