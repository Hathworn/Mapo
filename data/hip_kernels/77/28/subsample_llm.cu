#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    output += o * output_w * output_h;
    input += i * input_w * input_h;

    // Pre-calculate constant divisor
    float inv_kHW = 1.0f / float(kW * kH);

    // For all output pixels
    for (int yy = yy_start; yy < yy_end; yy += yy_step) {
        for (int xx = xx_start; xx < xx_end; xx += xx_step) {
            // Compute the mean of the input image
            float *ptr_input = input + yy * dH * input_w + xx * dW;
            float *ptr_output = output + yy * output_w + xx;
            float sum = 0.0f;

            // Unroll inner loops to optimize memory access and arithmetic operations
            #pragma unroll
            for (int ky = 0; ky < kH; ky++) {
                #pragma unroll
                for (int kx = 0; kx < kW; kx++) {
                    sum += ptr_input[kx];
                }
                ptr_input += input_w; // Move to the next input line
            }

            // Update output
            *ptr_output = sum * inv_kHW;  // Use precalculated division factor
        }
    }
}