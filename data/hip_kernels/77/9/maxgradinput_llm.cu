#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void maxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;
    
    // Determine current position in the grid
    int xx = blockIdx.x * blockDim.x + threadIdx.x;
    int yy = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure we are within the bounds of the output
    if (xx < output_w && yy < output_h) {
        // Calculate input/output offsets
        int o = blockIdx.z;
        int i = o;
        
        gradOutput += o * output_w * output_h;
        gradInput += i * input_w * input_h;
        indices_x += o * output_w * output_h;
        indices_y += o * output_w * output_h;
        
        // Calculate pointers
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float *ptr_ind_x = indices_x + yy * output_w + xx;
        float *ptr_ind_y = indices_y + yy * output_w + xx;
        
        // Read output and indices
        float z = *ptr_gradOutput;
        int argmax_x = (*ptr_ind_x) - 1;
        int argmax_y = (*ptr_ind_y) - 1;
        
        // Update gradInput
        float *ptr_gradInput = gradInput + argmax_y * input_w + argmax_x;
        atomicAdd(ptr_gradInput, z); // Use atomic operation to prevent race conditions
    }
}