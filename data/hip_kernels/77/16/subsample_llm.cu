#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y * gridDim.y;

    // select input/output plane
    output = output + o * output_w * output_h;
    input = input + i * input_w * input_h;

    // Get the good mask for (k,i) (k out, i in)
    float the_weight = weight[k];

    // Initialize to the bias
    float the_bias = bias[k];

    // Main loop over output dimensions
    for (int yy = yy_start; yy < yy_end; yy += yy_step) {
        for (int xx = xx_start; xx < xx_end; xx += xx_step) {
            // Pointers to input and output
            float *ptr_input = input + yy * dH * input_w + xx * dW;
            float *ptr_output = output + yy * output_w + xx;

            // Accumulate sum
            float sum = 0;
            for (int ky = 0; ky < kH; ky++) {
                #pragma unroll 4  // Unroll kernel width loop for better performance
                for (int kx = 0; kx < kW; kx++) {
                    sum += ptr_input[kx];
                }
                ptr_input += input_w; // next input line
            }
            // Update output with pre-computed weight and bias
            *ptr_output = the_weight * sum + the_bias;
        }
    }
}