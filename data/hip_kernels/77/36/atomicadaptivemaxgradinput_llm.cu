#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void atomicadaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w)
{
    // Compute global thread indices
    int xx = blockIdx.x * blockDim.x + threadIdx.x;
    int yy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (xx >= output_w || yy >= output_h) return;

    // Select input/output plane
    int o = blockIdx.z;
    int i = o;

    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // Compute starting positions
    int y_start = static_cast<int>(floor(float(yy) / output_h * input_h));
    int x_start = static_cast<int>(floor(float(xx) / output_w * input_w));

    float *ptr_gradInput = gradInput + y_start * input_w + x_start;
    float *ptr_gradOutput = gradOutput + yy * output_w + xx;
    float *ptr_ind_x = indices_x + yy * output_w + xx;
    float *ptr_ind_y = indices_y + yy * output_w + xx;
    float z = *ptr_gradOutput;

    int argmax_x = (*ptr_ind_x) - 1;
    int argmax_y = (*ptr_ind_y) - 1;

    // Atomic add since different threads could update the same variable
    atomicAdd(&(ptr_gradInput[argmax_x + argmax_y * input_w]), z);
}