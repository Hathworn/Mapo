#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_SoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int nframe, int dim)
{
    __shared__ float buffer[SOFTMAX_THREADS];
    int k = blockIdx.x;
    float *gradInput_k = gradInput + k * dim;
    float *output_k = output + k * dim;
    float *gradOutput_k = gradOutput + k * dim;

    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;

    // Initialize buffer and accumulate in a single pass
    float sum_k = 0.0f;
    for (int i = i_start; i < i_end; i += i_step)
    {
        sum_k += gradOutput_k[i] * output_k[i];
    }
    buffer[threadIdx.x] = sum_k;

    __syncthreads();

    // Optimized warp-level reduction for sum
    if (threadIdx.x < 32)
    {
        for (int offset = 32; offset < blockDim.x; offset += 32)
        {
            if (threadIdx.x + offset < blockDim.x)
            {
                buffer[threadIdx.x] += buffer[threadIdx.x + offset];
            }
        }

        sum_k = buffer[0];
        for (int offset = 16; offset > 0; offset /= 2)
        {
            sum_k += __shfl_down(sum_k, offset);
        }
        if (threadIdx.x == 0)
        {
            buffer[0] = sum_k;
        }
    }

    __syncthreads();

    sum_k = buffer[0];
    for (int i = i_start; i < i_end; i += i_step)
    {
        gradInput_k[i] = output_k[i] * (gradOutput_k[i] - sum_k);
    }
}