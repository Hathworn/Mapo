#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate thread indices
    int xx = blockIdx.x * blockDim.x + threadIdx.x;
    int yy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    if (xx < output_w && yy < output_h) 
    {
        // Select input and output plane
        int o = blockIdx.z; // Using 3rd dimension for multi-plane input/output
        float *ptr_gradOutput = gradOutput + o * output_w * output_h + yy * output_w + xx;
        float z = *ptr_gradOutput;

        float *ptr_gradInput = gradInput + o * input_w * input_h + yy * dH * input_w + xx * dW;
        
        // Compute gradInput with atomicAdd
        for (int ky = 0; ky < kH; ++ky) 
        {
            for (int kx = 0; kx < kW; ++kx) 
            {
                atomicAdd(&(ptr_gradInput[kx]), z / float(kW * kH)); // Atomic operation
            }
            ptr_gradInput += input_w;
        }
    }
}