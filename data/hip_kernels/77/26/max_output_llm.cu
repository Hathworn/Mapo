#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void max_output(float *input, float *output, float *indices, long nrows, long ncols) {
    // Calculate the global output index
    long o = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
    
    // Ensure output index is within the number of rows
    if (o >= nrows) return;

    // Compute input index offset directly using 'o'
    long i = o * ncols;
    
    // Set input pointer offset
    input += i;

    // Initialize max value and index.
    float max = input[0];
    long argmax = 0;

    // Unrolls the loop for better parallel execution
    for (long ii = 1; ii < ncols; ++ii) {
        float val = input[ii];
        if (val > max) {
            max = val;
            argmax = ii;
        }
    }

    // Store max value and its index in respective output arrays
    output[o] = max;
    indices[o] = argmax + 1; // Adjusting index for legacy reasons
}