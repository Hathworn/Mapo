#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinput(float *gradInput, float *gradOutput, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // get current thread positions in the block
    int o = blockIdx.x;
    int i = o;

    // calculate global index positions
    int xx = threadIdx.x + blockIdx.y * blockDim.x;
    int yy = threadIdx.y + blockIdx.z * blockDim.y;

    // check if xx, yy are within output bounds
    if (xx < output_w && yy < output_h) {

        // calculate pointers to the corresponding locations
        gradOutput = gradOutput + o * output_w * output_h;
        gradInput = gradInput + i * input_w * input_h;

        float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float z = *ptr_gradOutput;

        // update gradInput using shared data from gradOutput
        for (int ky = 0; ky < kH; ky++) {
            for (int kx = 0; kx < kW; kx++) {
                ptr_gradInput[kx] += z / float(kW * kH);
            }
            ptr_gradInput += input_w;
        }
    }
}