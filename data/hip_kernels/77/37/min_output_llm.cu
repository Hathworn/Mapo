#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void min_output(float *input, float *output, float *indices, long nrows, long ncols)
{
    // Calculate output offset
    long o = threadIdx.x + blockDim.x * blockIdx.x;
    if (o >= nrows) return;

    // Calculate input offset
    long i = o * ncols;

    // Initialize min and argmin with the first element
    float min = input[i];
    long argmin = 0;

    // Unroll loop to improve performance
    #pragma unroll
    for (long ii = 1; ii < ncols; ++ii) {
        float val = input[i + ii];  // Compute element address directly
        if (val < min) {
            min = val;
            argmin = ii;
        }
    }

    // Store results
    output[o] = min;
    indices[o] = argmin + 1;
}