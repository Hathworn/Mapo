#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_ClassNLLCriterion_updateOutput_kernel(float *output, float *input, float *target, int nframe, int ndim, int sizeAverage, int ntarget) {
    // Using shared memory and loop unrolling for optimization
    __shared__ float shInputs[NTHREADS]; 
    int idx = threadIdx.x;
    float sum = 0.0f;

    for (int i = idx; i < nframe; i += NTHREADS) {
        #pragma unroll // Unrolling for efficiency
        for (int j = 0; j < ntarget; ++j) {
            int t = static_cast<int>(target[i * ntarget + j]) - 1;
            if (t >= 0) {
                sum += input[i * ndim + t];
            }
        }
    }
    shInputs[idx] = sum;
    __syncthreads();

    if (idx == 0) {
        float total = 0.0f;
        // Unrolling final sum reduction for better performance
        for (int i = 0; i < NTHREADS; ++i) {
            total += shInputs[i];
        }
        if (sizeAverage) {
            total /= nframe;
        }
        *output = -total;
    }
}