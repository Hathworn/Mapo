#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Precompute indexes to minimize repeated calculations
    int time_idx = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    int gradInput_offset = blockIdx.x * input_w * input_n + time_idx * input_n * dW;
    int gradOutput_offset = blockIdx.x * output_w * input_n + time_idx * input_n;
    int indices_offset = gradOutput_offset;

    // Check valid output_w range
    if (time_idx < output_w) {
        // Optimize feature processing using pointer arithmetic
        float *gradInput_data_feat = gradInput + gradInput_offset;
        float *gradOutput_data_feat = gradOutput + gradOutput_offset;
        float *indices_data_feat = indices + indices_offset;

        // For all features using direct atomic access
        for (int feat = 0; feat < input_n; ++feat) {
            int index = (int)indices_data_feat[feat] * input_n + feat;
            atomicAdd(&gradInput_data_feat[index], gradOutput_data_feat[feat]);
        }
    }
}