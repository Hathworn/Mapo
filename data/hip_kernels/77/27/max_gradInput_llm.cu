#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void max_gradInput(float *input, float *output, float *indices, long nrows, long ncols)
{
    // Improve parallelism by processing multiple elements per thread
    long o = blockIdx.x * blockDim.x + threadIdx.x;
    long stride = blockDim.x * gridDim.x;

    for (; o < nrows; o += stride) {
        // input offset:
        long i = o * ncols;

        // bprop max gradient:
        long idx = indices[o] - 1;
        input[i + idx] = output[o];
    }
}