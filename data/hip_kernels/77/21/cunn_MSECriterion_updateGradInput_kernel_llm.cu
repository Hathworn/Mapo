#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_MSECriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, float norm, int nframe, int dim)
{
    int k = blockIdx.x;
    float *gradInput_k = gradInput + k * dim;
    float *input_k = input + k * dim;
    float *target_k = target + k * dim;

    int idx = blockDim.x * blockIdx.x + threadIdx.x;  // Global thread index

    // Ensure we do not exceed array boundaries
    if (idx < nframe * dim) {
        int i = idx % dim;
        gradInput_k[i] = norm * (input_k[i] - target_k[i]);
    }
}