#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void atomicmaxgradinput( float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW )
{
    // Obtain global thread indices for x and y
    int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    int global_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Check if the current thread index is within output bounds
    if (global_x < output_w && global_y < output_h) {
        // Calculate offsets for gradInput and gradOutput
        gradOutput += blockIdx.x * output_w * output_h;
        gradInput += blockIdx.x * input_w * input_h;
        indices_x += blockIdx.x * output_w * output_h;
        indices_y += blockIdx.x * output_w * output_h;

        // Pointers to the current processed elements
        float *ptr_gradInput = gradInput + global_y * dH * input_w + global_x * dW;
        float *ptr_gradOutput = gradOutput + global_y * output_w + global_x;
        float *ptr_ind_x = indices_x + global_y * output_w + global_x;
        float *ptr_ind_y = indices_y + global_y * output_w + global_x;

        // Read values
        float z = *ptr_gradOutput;

        // Calculate argmax indices
        int argmax_x = (*ptr_ind_x)-1;
        int argmax_y = (*ptr_ind_y)-1;

        // Atomic add for correct concurrent thread execution
        atomicAdd(&(ptr_gradInput[argmax_x + argmax_y * input_w]), z);
    }
}