#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradweight(float *input, float *gradOutput, float *gradWeight, float *gradBias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW, float scale)
{
    // Iterators
    int xx, yy;

    // Output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;

    int yy_start = threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    input += i * input_w * input_h;

    // Thread ID
    int tid = blockDim.x * threadIdx.y + threadIdx.x;

    // Create array to hold partial sums
    extern __shared__ float sums[];
    sums[tid] = 0.0f;

    // Compute partial sums
    for (yy = yy_start; yy < yy_end; yy += yy_step) {
        for (xx = xx_start; xx < xx_end; xx += xx_step) {
            float *ptr_input = input + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput;
            long kx, ky;
            for (ky = 0; ky < kH; ky++) {
                for (kx = 0; kx < kW; kx++) {
                    // Accumulate partial sums
                    sums[tid] += z * ptr_input[kx];
                }
                ptr_input += input_w;
            }
        }
    }
    __syncthreads();

    // Reduce: accumulate all partial sums to produce final gradWeight
    if (tid == 0) {
        for (int i = 0; i < blockDim.x * blockDim.y; i++) gradWeight[k] += scale * sums[i];
    }
    __syncthreads();

    // Compute gradBias
    sums[tid] = 0.0f;
    for (int i = tid; i < output_w * output_h; i += (blockDim.x * blockDim.y)) {
        sums[tid] += gradOutput[i];
    }
    __syncthreads();

    // Reduce gradBias
    if (tid == 0) {
        for (int i = 0; i < (blockDim.x * blockDim.y); i++)
            gradBias[k] += scale * sums[i];
    }
}