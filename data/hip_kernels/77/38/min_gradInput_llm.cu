#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void min_gradInput(float *input, float *output, float *indices, long nrows, long ncols)
{
    // Calculate the global index:
    long o = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Guard against out-of-bounds access:
    if (o < nrows) {
        // Compute input index and update gradient:
        long i = o * ncols;
        long idx = static_cast<long>(indices[o]) - 1; // Cast to handle fractional indices accurately
        input[i + idx] = output[o];
    }
}