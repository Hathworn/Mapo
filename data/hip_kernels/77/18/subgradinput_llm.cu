#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradinput(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Compute output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Calculate thread indices
    int xx = threadIdx.x + blockIdx.x * blockDim.x;
    int yy = threadIdx.y + blockIdx.y * blockDim.y;
    
    if(xx >= output_w || yy >= output_h) return; // Bounds check for threads

    // Compute batch and weight index
    int o = blockIdx.z;
    int i = o;
    int k = o % input_n;

    // Select input/output plane
    gradOutput = gradOutput + o * output_w * output_h;
    gradInput = gradInput + i * input_w * input_h;

    // Get weight
    float the_weight = weight[k];

    // Compute gradInput
    float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
    float *ptr_gradOutput = gradOutput + yy * output_w + xx;
    float z = *ptr_gradOutput * the_weight;

    // Use register for faster accesses
    float z_reg = z;

    // Loop over kernel height and width
    for(int ky = 0; ky < kH; ky++) {
        for(int kx = 0; kx < kW; kx++) {
            ptr_gradInput[kx] += z_reg; // Accumulate result in gradInput
        }
        ptr_gradInput += input_w; // Move to the next row
    }
}