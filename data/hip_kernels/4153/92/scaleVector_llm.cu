#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Calculate global thread index
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread operates within bounds
    if (pos < len) 
    {
        // Perform scaling operation
        d_res[pos] = d_src[pos] * scale;
    }
}