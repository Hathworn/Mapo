#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst) {
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Move common calculations out of conditional blocks
    float cx = u[flow_row_offset + j] * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + static_cast<float>(i) + 1.0f;
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    float value = src[image_row_offset + j];

    // Precompute weights
    float w_br = dx * dy;
    float w_bl = (1.0f - dx) * dy;
    float w_ul = (1.0f - dx) * (1.0f - dy);
    float w_ur = dx * (1.0f - dy);

    // Consolidate boundary checks to improve efficiency
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);

    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        _atomicAdd(dst + ty * image_stride + tx, value * w_br);
        _atomicAdd(normalization_factor + ty * image_stride + tx, w_br);
    }

    if (tx - 1 >= 0 && tx - 1 < w && ty >= 0 && ty < h) {
        _atomicAdd(dst + ty * image_stride + (tx - 1), value * w_bl);
        _atomicAdd(normalization_factor + ty * image_stride + (tx - 1), w_bl);
    }

    if (tx - 1 >= 0 && tx - 1 < w && ty - 1 >= 0 && ty - 1 < h) {
        _atomicAdd(dst + (ty - 1) * image_stride + (tx - 1), value * w_ul);
        _atomicAdd(normalization_factor + (ty - 1) * image_stride + (tx - 1), w_ul);
    }

    if (tx >= 0 && tx < w && ty - 1 >= 0 && ty - 1 < h) {
        _atomicAdd(dst + (ty - 1) * image_stride + tx, value * w_ur);
        _atomicAdd(normalization_factor + (ty - 1) * image_stride + tx, w_ur);
    }
}