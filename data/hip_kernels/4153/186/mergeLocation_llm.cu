#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Acquire the point index for this thread
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform computation only if the point index is within bounds
    if (ptidx < npoints)
    {
        // Load data from global memory to register (faster access)
        short2 loc = loc_[ptidx];

        // Perform scaling operations
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}