#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate unique thread index within the grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure indices are within matrix boundaries
    if (idx < w && idy < h) {
        const int pos = idy * w + idx; // Compute linear position for 2D array
        image[pos] = value; // Set the value
    }
}