#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < h && j < w)  // Simplified boundary condition check
    {
        int flow_row_offset = i * flow_stride;
        int image_row_offset = i * image_stride;

        float u_ = u[flow_row_offset + j];
        float v_ = v[flow_row_offset + j];

        // Compute the target pixel positions
        float cx = u_ * time_scale + (float)j + 1.0f;
        float cy = v_ * time_scale + (float)i + 1.0f;

        // Pixel location with rounding
        int tx = __float2int_rn(cx);
        int ty = __float2int_rn(cy);

        float value = src[image_row_offset + j];

        // Check boundary and perform atomic addition
        if (tx >= 0 && tx < w && ty >= 0 && ty < h)  // Adjusted boundary checks
        {
            atomicAdd(&dst[ty * image_stride + tx], value);
        }
    }
}