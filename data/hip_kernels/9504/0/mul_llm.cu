#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int N = 32;

__global__ void mul(int* A, int* B, int* C) {
    // Cache thread index
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int lig = blockIdx.y * blockDim.y + threadIdx.y;

    // Cache the current index of the result matrix
    int index = lig * N + col;

    if (col < N && lig < N) {
        int inter = 0;

        // Loop with stride to improve memory access pattern
        for (int i = threadIdx.x; i < N; i += blockDim.x) {
            inter += A[lig * N + i] * B[i * N + col];
        }

        C[index] += inter; // Ensure atomicity by accumulating results
    }
}