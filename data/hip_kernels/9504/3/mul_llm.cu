#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul(int* A, int* B, int* C) {
    // Calculate row and column indices using block and thread indices.
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Combine the two conditions into a single if statement to check valid thread processing.
    if (col < N && row < N) {
        int result = 0;
        // Unroll the loop for better performance when possible.
        // Use registers for temporary variable 'result' to reduce memory access overhead.
        #pragma unroll
        for (int i = 0; i < N; ++i) {
            result += A[row * N + i] * B[i * N + col];
        }
        // Set result in output matrix C.
        C[row * N + col] = result;
    }
}