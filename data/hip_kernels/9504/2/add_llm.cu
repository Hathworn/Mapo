#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    // Use thread index within block to leverage more parallelism
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Ensure we do not access out-of-bound array elements
    if (idx < gridDim.x * blockDim.x) {
        c[idx] = a[idx] + b[idx];
    }
}