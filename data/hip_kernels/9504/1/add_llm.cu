#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    // Use blockIdx.x, blockDim.x and threadIdx.x for grid-stride loop
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int gridSize = blockDim.x * gridDim.x;

    // Stride through the arrays in case number of threads is less than array size
    while (index < N) {
        c[index] = a[index] + b[index];
        index += gridSize;
    }
}