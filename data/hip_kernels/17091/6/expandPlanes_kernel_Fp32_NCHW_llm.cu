#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expandPlanes_kernel_Fp32_NCHW(float* output, const uint64_t* masks, const float* values, int n) {
    // Block size of 256, same mask/val for 64 consecutive threads.
    constexpr int kNumShmemElements = 256 / 64;

    __shared__ uint64_t shMasks[kNumShmemElements];
    __shared__ float shVals[kNumShmemElements];

    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int planeIndex = index >> 6;

    // Exit if index is out of bounds
    if (planeIndex >= n) return;

    // Load inputs to shared memory only with necessary threads
    if (threadIdx.x < 64) {
        int shIndex = threadIdx.x >> 6;
        shMasks[shIndex] = masks[planeIndex + shIndex];
        shVals[shIndex] = values[planeIndex + shIndex];
    }
    __syncthreads();

    // Read mask and value from shared memory
    uint64_t mask = shMasks[threadIdx.x >> 6];
    float op = shVals[threadIdx.x >> 6];

    // Compute output based on mask
    output[index] = (mask & (1ull << (index & 0x3F))) ? op : 0.0f;
}
```
