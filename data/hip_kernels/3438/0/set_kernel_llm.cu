#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_kernel(int* dst, int const value, int const count)
{
    int const index = blockIdx.x * blockDim.x + threadIdx.x;

    // Early return to minimize unnecessary threads from executing further
    if (index < count)
    {
        dst[index] = value;
    }
}