#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_kernel(REAL* dst, REAL const value, std::size_t const count)
{
    // Use shared memory for faster access
    extern __shared__ REAL sharedValue[];

    // Load value into shared memory
    if (threadIdx.x == 0) {
        sharedValue[0] = value;
    }
    __syncthreads();

    std::size_t const index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use branchless approach
    if (index < count) {
        dst[index] = sharedValue[0];
    }
}