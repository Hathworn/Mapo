#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define N 1024*4

// Device Kernel
__global__ void amean(float *A, float *S)
{
    // Use shared memory for efficient data reduction
    extern __shared__ float sdata[];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    sdata[tid] = A[i];
    __syncthreads();

    // Perform reduction using less synchronization
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        S[blockIdx.x] = sdata[0];
    }
}