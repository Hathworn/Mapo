#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access and computation efficiency
__global__ void PreProcessVectorFieldDev(float3* field, float width, float height, float minx, float miny)
{
    // Calculate global thread index
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Fetch field elements and compute offsets
    float3 p1 = field[idx * 2 + 0];
    float3 p2 = field[idx * 2 + 1];

    // Reuse computation results for efficiency
    float minx_sub = minx;
    float miny_sub = miny;
    float width_div = 1.0f / width;
    float height_div = 1.0f / height;

    // Process point 1
    p1.x = (p1.x - minx_sub) * width_div;
    p1.y = (p1.y - miny_sub) * height_div;
    p1.z = 0; // Assign zero directly as there's no computation

    // Process point 2
    p2.x = (p2.x - minx_sub) * width_div;
    p2.y = (p2.y - miny_sub) * height_div;
    p2.z = 0; // Assign zero directly as there's no computation

    // Write back processed data
    field[idx * 2 + 0] = p1;
    field[idx * 2 + 1] = p2;
}