#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SampleConcentrationDev(float* concentration, const uint2* cellStartEnd) {
    // Calculate the linear index of the current thread
    const uint cellid = blockIdx.x + blockIdx.y * gridDim.x + threadIdx.x * gridDim.x * gridDim.y;
    
    // Directly load the start and end values for the cell
    uint2 cellStEnd = cellStartEnd[cellid];

    // Store the difference (concentration) in the output array
    concentration[cellid] = cellStEnd.y - cellStEnd.x;
}