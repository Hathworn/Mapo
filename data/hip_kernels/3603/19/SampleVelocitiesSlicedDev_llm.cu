#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float length(float3 vec) {
    return sqrtf(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
}

__device__ float length4(float4 vec) {
    return sqrtf(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
}

__global__ void SampleVelocitiesSlicedDev(float* velocities, const uint slice, const float4* vels_data, const uint2* cellStartEnd, const uint* indices) {
    const uint cellid = gridDim.x * blockDim.x * slice + threadIdx.x * gridDim.x + blockIdx.x;
    uint2 cellStEnd = cellStartEnd[cellid];
    const uint part_in_cell = cellStEnd.y - cellStEnd.x;

    if (part_in_cell > 0) {
        float4 p = make_float4(0, 0, 0, 0);

        for (uint index = cellStEnd.x; index < cellStEnd.y; index++) {
            #ifndef REORDER
            uint idx = indices[index];
            float4 vel = vels_data[idx];
            #else
            float4 vel = vels_data[index];
            #endif

            p.x += vel.x;
            p.y += vel.y;
            p.z += vel.z;
        }

        // Compute velocity length and store result
        velocities[threadIdx.x * gridDim.x + blockIdx.x] = length4(p) / part_in_cell;
    } else {
        // No particles in cell, set velocity to 0
        velocities[threadIdx.x * gridDim.x + blockIdx.x] = 0.0f;
    }
}