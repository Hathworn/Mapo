#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BuildColorFieldDev(float* data, uchar4* colors, float* minmax, uint xx, uint yy) 
{
    float mn = minmax[0];
    float mx = minmax[1];
    float median = (mx - mn) * 0.5f;

    // Calculate a unique global index for the current thread
    const uint idx = blockIdx.x * blockDim.x + threadIdx.x; 

    float val = data[idx];
    uchar4 col;

#if 1
    // Use ternary operators for simplification
    float alpha = (val < median) ? (val - mn) / (median - mn) : (val - median) / (mx - median);
    
    col.x = (val < median) ? 0 : 255 * alpha;
    col.y = (val < median) ? 255 * (1 - alpha) : 0;
    col.z = (val < median) ? 255 * alpha : 255 * (1 - alpha);
#else
    float alpha = 1;

    if (!(val < 0.1 || mn == mx || mx < 0.1))
        alpha = val / (mx - mn);

    col.x = col.y = col.z = 255 * (1 - alpha);
#endif
    col.w = 255;

    // Update the global memory for color based on unique color index
    const uint col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    colors[col_idx] = col;
}