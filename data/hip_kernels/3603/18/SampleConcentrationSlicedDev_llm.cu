#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SampleConcentrationSlicedDev(float* concentration, uint slice, const uint2* cellStartEnd)
{
    // Calculate unique thread index for the block
    const uint cellid = blockIdx.x + blockDim.x * (threadIdx.x + slice * blockDim.x);
    
    // Load cell start and end indices
    uint2 cellStEnd = cellStartEnd[cellid];

    // Write the concentration result back to global memory
    concentration[cellid] = cellStEnd.y - cellStEnd.x; // Directly use 'cellid' for global write
}