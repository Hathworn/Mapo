#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "SampleConcentrationSlicedDev_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *concentration = NULL;
hipMalloc(&concentration, XSIZE*YSIZE);
uint slice = 1;
const uint2 *cellStartEnd = NULL;
hipMalloc(&cellStartEnd, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
SampleConcentrationSlicedDev<<<gridBlock, threadBlock>>>(concentration,slice,cellStartEnd);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
SampleConcentrationSlicedDev<<<gridBlock, threadBlock>>>(concentration,slice,cellStartEnd);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
SampleConcentrationSlicedDev<<<gridBlock, threadBlock>>>(concentration,slice,cellStartEnd);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}