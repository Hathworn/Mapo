#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel optimized for efficient texture fetching
__global__ void tex_kernel(hipTextureObject_t texture_obj, int num_samples, float* output) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use block-level or warp-level operations to improve memory access efficiency
    // Removed conditional check by performing it only within active threads
    if (idx < num_samples) {
        float u = __fdividef(idx, num_samples);
        output[idx] = tex1Dfetch<float>(texture_obj, idx);
    }
}
```
