#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void postprocess_matrix(float* matrix, long* long_indices, int* indices, unsigned int N_POINTS, unsigned int K)
{
    // Calculate global thread ID
    int TID = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure TID is within bounds
    if (TID < N_POINTS * K) {
        // Directly transfer the value from long_indices to indices
        indices[TID] = static_cast<int>(long_indices[TID]);
    }
}