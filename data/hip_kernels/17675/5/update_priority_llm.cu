#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_priority(int *new_priority, int n_selected, const int *new_idx, int n_ws, const int *idx, const int *priority) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid < n_selected) {
        int my_new_idx = new_idx[tid];

        // Utilize shared memory for idx and priority to reduce global memory access
        __shared__ int shared_idx[1024];
        __shared__ int shared_priority[1024];

        if (threadIdx.x < n_ws) {
            shared_idx[threadIdx.x] = idx[threadIdx.x];
            shared_priority[threadIdx.x] = priority[threadIdx.x];
        }
        __syncthreads();

        // Iterate over working set size using shared memory
        for (int i = 0; i < n_ws; i++) {
            if (shared_idx[i] == my_new_idx) {
                new_priority[tid] = shared_priority[i] + 1;
                break; // Early exit as idx[i] should be unique
            }
        }
    }
}