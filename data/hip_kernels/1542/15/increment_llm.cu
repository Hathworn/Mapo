#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void increment(char* data, size_t length)
{
    // Optimize memory access by using a block-local variable for length
    size_t global_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (global_index < length) {
        // Ensure coalesced memory access by accessing data by global_index
        data[global_index]++;
    }
}