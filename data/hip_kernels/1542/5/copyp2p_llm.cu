#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyp2p(int4* __restrict__ dest, const int4* __restrict__ src, size_t num_elems)
{
    size_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
    size_t gridSize = blockDim.x * gridDim.x;

    // Use loop unrolling and restrict keyword for performance optimization
    #pragma unroll
    for (size_t i = globalId; i < num_elems; i += gridSize)
    {
        dest[i] = src[i];
    }
}