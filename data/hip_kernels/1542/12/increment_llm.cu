#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void increment(char* data, size_t length)
{
    // Use a for loop to allow each thread to handle multiple elements
    size_t global_index = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = global_index; i < length; i += stride)
    {
        data[i]++;
    }
}