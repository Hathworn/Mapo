#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "from_2D_texture_to_memory_space_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    hipTextureObject_t texture_source = 1;
float *destination = NULL;
hipMalloc(&destination, XSIZE*YSIZE);
size_t w = XSIZE;
size_t h = YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
from_2D_texture_to_memory_space<<<gridBlock, threadBlock>>>(texture_source,destination,w,h);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
from_2D_texture_to_memory_space<<<gridBlock, threadBlock>>>(texture_source,destination,w,h);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
from_2D_texture_to_memory_space<<<gridBlock, threadBlock>>>(texture_source,destination,w,h);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}