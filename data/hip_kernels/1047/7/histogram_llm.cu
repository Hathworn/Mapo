#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogram (unsigned char *utime, unsigned int* histo, size_t n)
{
    __shared__ unsigned int lhisto[512];

    // Initialize shared memory
    lhisto[threadIdx.x] = 0;
    __syncthreads();

    int globalIndex = threadIdx.x + blockIdx.x * blockDim.x;

    // Optimization: combine two separate loops into one for simplicity
    for (int i = globalIndex; i < n; i += blockDim.x * gridDim.x) {
        int offset = (i < n / 2) ? 0 : 256;
        atomicAdd(&lhisto[utime[i] + offset], 1);
    }
    __syncthreads();

    // Global accumulation
    atomicAdd(&histo[threadIdx.x], lhisto[threadIdx.x]);
}