#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void digitize(float* idat, uint8_t* udat, size_t n)
{
    // Calculate the global thread ID
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use a stride loop to cover all elements
    int stride = blockDim.x * gridDim.x;
    
    for (; i < n; i += stride)
    {
        // Efficient normalization calculation
        float tmp = idat[i] * 16.9096462f + 128.5f;

        // Simplified clamping
        tmp = (tmp < 0.0f) ? 0.0f : ((tmp > 255.0f) ? 255.0f : tmp);
        
        // Direct casting to uint8_t
        udat[i] = (uint8_t)tmp;
    }
}