#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simpleKernel(float *dst, const float *src1, const float *src2)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Perform the addition only if idx is within bounds for better safety
    if (idx < gridDim.x * blockDim.x) {
        dst[idx] = src1[idx] + src2[idx];
    }
}