#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simpleKernel(float *dst, float *src)
{
    // Use shared memory for better memory access efficiency
    __shared__ float shared_src[256]; // Assuming blockDim.x is 256

    // Load data into shared memory
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    shared_src[threadIdx.x] = src[idx];
    __syncthreads();

    // Calculate square using shared memory
    float temp = shared_src[threadIdx.x];
    dst[idx] = temp * temp;
}