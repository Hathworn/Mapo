#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Efficient division using reciprocal
    const float reciprocal_N = 1.0f / N;
    *accuracy *= reciprocal_N;
}