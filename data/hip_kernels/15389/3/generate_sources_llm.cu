#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_sources(hiprandState *state, int n, uint32_t *verts) {
    // Calculate unique global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (id < n) {
        // Use local state for generating random numbers
        hiprandState local_state = state[id];
        for (int idx = id; idx < n; idx += stride) {
            verts[idx] = hiprand(&local_state);
        }
        // Update global state
        state[id] = local_state;
    }
}