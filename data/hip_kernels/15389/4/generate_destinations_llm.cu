#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel to generate destinations
__global__ void generate_destinations(hiprandState *state, int n, const uint32_t *sources, uint32_t *destinations) {
    int first = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    hiprandState local_state = state[first];
    
    for (int id = first; id < n; id += stride) {
        // Reduce modulo operations
        uint32_t random_index = hiprand(&local_state);
        random_index = random_index >= n ? random_index % n : random_index;
        destinations[id] = sources[random_index];
    }

    state[first] = local_state;  // Persist local state
}