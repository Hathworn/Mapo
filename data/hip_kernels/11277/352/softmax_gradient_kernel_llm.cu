#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel( const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    
    // Initialize shared memory to zero
    float sum_dY_Y = 0.0f;

    // Perform reduction within each block
    for (int i = idx; i < dim; i += blockDim.x) {
        sum_dY_Y += dY[i] * Y[i];
    }
    reduction_buffer[idx] = sum_dY_Y;
    __syncthreads();

    // Reduce the results from each thread to a single value in the block's shared memory
    if (idx == 0) {
        float block_sum = 0.0f;
        for (int i = 0; i < blockDim.x; ++i) {
            block_sum += reduction_buffer[i];
        }
        reduction_buffer[0] = block_sum;
    }
    __syncthreads();

    // Compute the gradient using the reduced sum
    float tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}