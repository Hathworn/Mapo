#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float temp_sum = 0.0f; // Use register for accumulation

    // Unroll inner loop
    if (eidx < numElements) {
        const uint stepSize = gridDim.x * DP_BLOCKSIZE;
        for (; eidx < numElements; eidx += stepSize) {
            temp_sum += a[eidx] * b[eidx];
        }
    }
    shmem[threadIdx.x] = temp_sum; // Store sum in shared memory
    __syncthreads();

    // Unroll reduction loop
    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    __syncthreads();
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem; // Eliminate bank conflicts
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    // Write result
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}