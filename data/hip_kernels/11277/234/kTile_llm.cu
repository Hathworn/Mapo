#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate indices once to reduce redundant computation
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    const uint totalElements = tgtWidth * tgtHeight;
    
    // Loop with stride for each thread to cover all elements
    for (uint i = idx; i < totalElements; i += numThreads) {
        // Compute output coordinates
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        
        // Effective input coordinates using modulo operation
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        
        // Copy data from source to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}