#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim.x for better flexibility

    if (tx < numCases) {
        const int label = static_cast<int>(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Initialize numMax to 0 before entering the loop
        if (labelp == maxp) { // Simplify the conditional check
            int numMax = 0;
            // Use shared memory to increase memory access speed
            __shared__ float sharedProbs[LOGREG_ERR_THREADS_X];

            for (int i = threadIdx.x; i < numOut; i += blockDim.x) { // Unroll the loop for better performance
                sharedProbs[i % LOGREG_ERR_THREADS_X] = probs[i * numCases + tx];
                __syncthreads(); // Ensure all threads have stored the current iteration's value
                numMax += (sharedProbs[i % LOGREG_ERR_THREADS_X] == maxp);
                __syncthreads(); // Ensure computation is done on all values from current iteration
            }
            correctProbs[tx] = 1.0f / float(numMax);
        } else {
            correctProbs[tx] = 0.0f; // Use 0.0f for float consistency
        }
    }
}