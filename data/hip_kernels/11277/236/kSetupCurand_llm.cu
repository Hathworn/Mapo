#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    // Optimize by ensuring the operation is only performed for valid thread indices
    if(tidx < NUM_RND_THREADS_PER_BLOCK * gridDim.x) {
        hiprand_init(seed, tidx, 0, &state[tidx]);
    }
}