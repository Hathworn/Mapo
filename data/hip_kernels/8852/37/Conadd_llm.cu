#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved version of Conadd kernel
__global__ void Conadd(float* a, float* b, float* c, int n) {
    // Compute the global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Check bounds and perform the addition
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}