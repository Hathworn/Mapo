#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Compute unique global index for each thread
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIdy = blockIdx.y * blockDim.y + threadIdx.y;

    // Flattened global position for efficiency
    int pos = globalIdy * w + globalIdx;

    if (globalIdy < h && globalIdx < w)
    {
        image[pos] = value; // Direct memory set for active threads
    }
}