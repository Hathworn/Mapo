#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Use a strided loop for better memory coalescing and handling larger data sizes
    for (int pos = blockIdx.x * blockDim.x + threadIdx.x; pos < len; pos += blockDim.x * gridDim.x) {
        d_res[pos] = d_src[pos] * scale;
    }
}