#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Ensure thread is within image bounds
    if (i < h && j < w) {
        const int pos = i * s + j;

        float scale = normalization_factor[pos];

        // Use ternary operator for inversion, avoiding division by zero
        float invScale = (scale == 0.0f) ? 1.0f : __frcp_rn(scale);

        // Perform normalization
        image[pos] *= invScale;
    }
}