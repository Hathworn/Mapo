#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate unique thread index
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Process only valid points
    if (ptidx < npoints)
    {
        // Read the input location
        short2 loc = loc_[ptidx];

        // Perform scaling
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}