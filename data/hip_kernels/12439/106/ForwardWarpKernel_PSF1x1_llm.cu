#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst) {
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Compute corner position with time scaling
    float cx = u_ * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v_ * time_scale + static_cast<float>(i) + 1.0f;
    
    // Convert to integer coordinates
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Optimize memory access by precalculating value index
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0)) {
        int dst_index = ty * image_stride + tx;
        float value = src[image_row_offset + j];
        // Use atomic operation to handle concurrent writes
        atomicAdd(dst + dst_index, value);
    }
}