#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(float *a, float *b, float *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique thread index

    if (tid < N) { // Ensure index is within bounds
        c[tid] = a[tid] + b[tid]; // Perform addition
    }
}