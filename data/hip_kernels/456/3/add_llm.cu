#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c) {
    // Compute only for a single thread to avoid race conditions
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *c = a + b;
    }
}