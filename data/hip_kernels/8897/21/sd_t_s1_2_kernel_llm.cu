#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 256  // Define a block size for better parallelization

__global__ void sd_t_s1_2_kernel(size_t h1d, size_t h2d, size_t h3d, size_t p4d, size_t p6d, 
                                 size_t p4ld_t2, size_t h1ld_t2, size_t h3ld_v2, size_t h2ld_v2, 
                                 size_t p6ld_v2, size_t h3ld_t3, size_t h2ld_t3, size_t h1ld_t3, 
                                 size_t p6ld_t3, size_t p4ld_t3, double *t2_d, double *v2_d, 
                                 size_t p4, size_t total_x, double *t3d) {
    size_t h1, h2, h3, p6;
    __shared__ double t2_shm[T1 * 4 * Tcomm];

    // Load data into shared memory
    for (size_t i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        if (i < h1d * p4d) {
            t2_shm[i] = t2_d[i];
        }
    }
    
    __syncthreads();  // Ensure shared memory is fully populated

    size_t thread_x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = gridDim.x * blockDim.x;  // Stride for grid parallelism

    // Optimize loop for grid-stride approach
    for (size_t i = thread_x; i < total_x; i += stride) {
        size_t rest_x = i;
        h3 = rest_x % h3d;
        rest_x /= h3d;
        h2 = rest_x % h2d;
        rest_x /= h2d;
        p6 = rest_x % p6d;

        // Loop over h1 and p4 dimensions
        for (h1 = 0; h1 < h1d; h1++) {
            for (p4 = 0; p4 < p4d; p4++) {
                // Perform the calculation
                t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4 * p4ld_t3] -=
                    t2_shm[h1 * p4d + p4] * v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2];
            }
        }
    }
}