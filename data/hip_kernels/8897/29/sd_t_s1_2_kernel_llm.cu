#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_2_kernel(int h1d, int h2d, int h3d, int p4d, int p6d, int p4ld_t2, int h1ld_t2, int h3ld_v2, int h2ld_v2, int p6ld_v2, int h3ld_t3, int h2ld_t3, int h1ld_t3, int p6ld_t3, int p4ld_t3, double *t2_d, double *v2_d, int p4, int total_x, double *t3d) {
    int h1, h2, h3, p6;
    __shared__ double t2_shm[T1 * 2 * Tcomm];

    // Load t2 to shared memory in a coalesced way
    for (int i = threadIdx.x; i < h1d * p4d; i += blockDim.x)
        if (i < h1d * p4d)
            t2_shm[i] = t2_d[i];

    int thread_x = T2 * T1 * blockIdx.x + threadIdx.x;
    __syncthreads();
    
    int start_idx = thread_x; // Start index for this thread in global execution
    int max_idx = total_x * gridDim.x; // Upper bound for index computation

    for (int idx = start_idx; idx < max_idx; idx += gridDim.x * blockDim.x) {
        int linear_idx = idx < total_x ? idx : total_x - 1;

        // Compute h3, h2, p6 efficiently
        h3 = linear_idx % h3d;
        linear_idx /= h3d;
        h2 = linear_idx % h2d;
        linear_idx /= h2d;
        p6 = linear_idx % p6d;

        if (idx < total_x) {
            for (h1 = 0; h1 < h1d; h1++) {
                for (p4 = 0; p4 < p4d; p4++) {
                    t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4 * p4ld_t3] -=
                        t2_shm[h1 * p4d + p4] * v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2];
                }
            }
        }
    }
    __syncthreads();
}