#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_6_kernel(size_t h1d,size_t h2d,size_t h3d,size_t p4d,size_t p5d,size_t p6d,size_t p4ld_t2,size_t h1ld_t2,size_t h3ld_v2,size_t h2ld_v2,size_t p6ld_v2,size_t p5ld_v2,size_t h3ld_t3,size_t h2ld_t3,size_t h1ld_t3,size_t p6ld_t3,size_t p5ld_t3,size_t p4ld_t3,double *t3d, double *t2_d, double *v2_d,size_t total_x) {
    size_t h1,h2,h3,p6,p5;
    __shared__ double t2_shm[T1*4*Tcomm];

    // Efficiently load shared memory with necessary data
    for(size_t i=threadIdx.x; i < h1d*p4d; i += blockDim.x)
        if(i < h1d*p4d)
            t2_shm[i] = t2_d[i];
    __syncthreads();
    
    size_t rest_x=blockIdx.x;
    size_t thread_x = T2*T1 * rest_x + threadIdx.x;

    // Loop refactored to minimize repetitive calculations
    for(size_t i=thread_x; i < total_x; i+=blockDim.x*gridDim.x) {

        size_t temp = i;
        h3 = temp % h3d;
        temp /= h3d;
        h2 = temp % h2d;
        temp /= h2d;
        p6 = temp % p6d;
        temp /= p6d;
        p5 = temp % p5d;

        for(h1=0; h1<h1d; ++h1) {
            for(p4=0; p4<p4d; ++p4) {
                // Simplified memory access within the innermost loop for efficiency
                double t2_val = t2_shm[h1*p4d + p4];
                atomicAdd(&t3d[h3*h3ld_t3 + h2*h2ld_t3 + h1*h1ld_t3 + p6*p6ld_t3 + p5*p5ld_t3 + p4*p4ld_t3], 
                          -t2_val * v2_d[h3*h3ld_v2 + h2*h2ld_v2 + p6*p6ld_v2 + p5*p5ld_v2]);
            }
        }
    }
    __syncthreads();
}