#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_6_kernel(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d, int p4ld_t2, int h1ld_t2, int h3ld_v2, int h2ld_v2, int p6ld_v2, int p5ld_v2, int h3ld_t3, int h2ld_t3, int h1ld_t3, int p6ld_t3, int p5ld_t3, int p4ld_t3, double *t3d, double *t2_d, double *v2_d, int p4, int total_x) {
    int h1, h2, h3, p6, p5;
    __shared__ double t2_shm[T1*2*Tcomm];

    // Load t2 into shared memory more efficiently
    for(int i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        t2_shm[i] = t2_d[i];
    }
    __syncthreads();

    int rest_x = blockIdx.x * T2 * T1 + threadIdx.x; // Calculate global thread index
    int thread_x;
    
    // Eliminate unnecessary rest_x calculations by bringing them inside the loop
    for(int i = 0; i < total_x; i += gridDim.x * blockDim.x) {
        thread_x = rest_x + i; // Update thread's execution context

        if(thread_x < total_x) {
            int tmp = thread_x;
            h3 = tmp % h3d; tmp /= h3d;
            h2 = tmp % h2d; tmp /= h2d;
            p6 = tmp % p6d; tmp /= p6d;
            p5 = tmp % p5d;

            for(h1 = 0; h1 < h1d; h1++) {
                for(p4 = 0; p4 < p4d; p4++) {
                    // Access shared memory (t2_shm) and global memory (v2_d) for t3d update
                    t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p5 * p5ld_t3 + p4 * p4ld_t3] -=
                        t2_shm[h1 * p4d + p4] * v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2 + p5 * p5ld_v2];
                }
            }
        }
    }
    __syncthreads();
}