#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_1_kernel(size_t h1d, size_t h2d, size_t h3d, size_t p4d, size_t p6d, size_t p4ld_t2,
                                 size_t h1ld_t2, size_t h3ld_v2, size_t h2ld_v2, size_t p6ld_v2, size_t h3ld_t3,
                                 size_t h2ld_t3, size_t h1ld_t3, size_t p6ld_t3, size_t p4ld_t3, double *t2_d,
                                 double *v2_d, size_t p4, size_t total_x, double* t3d) {
    size_t h1, h2, h3, p6;
    __shared__ double t2_shm[T1 * 4 * Tcomm];

    // Load t2_d into shared memory
    size_t t2_share_idx = threadIdx.x;
    while (t2_share_idx < h1d * p4d) {
        t2_shm[t2_share_idx] = t2_d[t2_share_idx];
        t2_share_idx += blockDim.x;
    }
    __syncthreads();

    size_t rest_x = blockIdx.x;
    size_t thread_x = T2 * T1 * rest_x + threadIdx.x;
    rest_x = thread_x;

    for (size_t i = 0; i < total_x; i += gridDim.x * blockDim.x) {
        size_t current_x = thread_x + i;
        if (current_x >= total_x) break;

        rest_x = current_x;
        h3 = rest_x % h3d;
        rest_x /= h3d;
        h2 = rest_x % h2d;
        rest_x /= h2d;
        p6 = rest_x % p6d;

        // Utilize shared memory for t2_shm in the computation
        for (h1 = 0; h1 < h1d; h1++) {
            for (p4 = 0; p4 < p4d; p4++) {
                t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4 * p4ld_t3] += 
                t2_shm[h1 * p4d + p4] * v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2];
            }
        }
    }
    __syncthreads();
}