#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_7_kernel(size_t h1d,size_t h2d,size_t h3d,size_t p4d,size_t p6d,size_t p4ld_t2,size_t h1ld_t2,size_t h3ld_v2,size_t h2ld_v2,size_t p6ld_v2,size_t h3ld_t3,size_t h2ld_t3,size_t h1ld_t3,size_t p6ld_t3,size_t p4ld_t3,double *t3d, double *t2_d, double *v2_d,size_t p4, size_t total_x) {
    size_t h1, h2, h3, p6;
    __shared__ double t2_shm[T1 * 4 * Tcomm];

    // Efficient block-wide memory copy to shared memory
    for (size_t i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        if (i < h1d * p4d) {
            t2_shm[i] = t2_d[i];
        }
    }
    __syncthreads();

    size_t rest_x = blockIdx.x;
    size_t thread_x = T2 * T1 * rest_x + threadIdx.x;

    for (size_t i = thread_x; i < total_x; i += gridDim.x * blockDim.x) {
        size_t ix = i;  // Use local copy of loop index

        // Unrolled compound assignments for rest_x decomposition
        h3 = ix % h3d; ix /= h3d;
        h2 = ix % h2d; ix /= h2d;
        p6 = ix % p6d;

        for (h1 = 0; h1 < h1d; ++h1) {
            for (p4 = 0; p4 < p4d; ++p4) {
                // Optimize by using registers for frequently accessed indexes
                double t2_val = t2_shm[h1 * p4d + p4];
                double v2_val = v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2];
                
                // Accumulate results into global memory
                atomicAdd(&t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4 * p4ld_t3], 
                          t2_val * v2_val);
            }
        }
    }
}
```
