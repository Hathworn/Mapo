#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sd_t_s1_8_kernel(int h1d, int h2d, int h3d, int p4d, int p6d, int p4ld_t2, int h1ld_t2, int h3ld_v2, int h2ld_v2, int p6ld_v2, int h3ld_t3, int h2ld_t3, int h1ld_t3, int p6ld_t3, int p4ld_t3, double *t3d, double *t2_d, double *v2_d, int p4, int total_x) {
    int h1, h2, h3, p6;
    __shared__ double t2_shm[T1*2*Tcomm];

    // Load t2_d into shared memory
    for (int i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        if (i < h1d * p4d)
            t2_shm[i] = t2_d[i];
    }
    __syncthreads();

    int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop across work, handle thread offset
    for (int rest_x = thread_x; rest_x < total_x; rest_x += stride) {
        int tmp = rest_x;
        h3 = tmp % h3d;
        tmp /= h3d;
        h2 = tmp % h2d;
        tmp /= h2d;
        p6 = tmp % p6d;

        // Compute t3d by utilizing t2_shm (in shared memory) and v2_d
        for (h1 = 0; h1 < h1d; h1++) {
            for (int local_p4 = 0; local_p4 < p4d; local_p4++) { // Renamed loop variable to avoid name conflict
                t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + local_p4 * p4ld_t3] -= 
                    t2_shm[h1 * p4d + local_p4] * 
                    v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2];
            }
        }
    }
    __syncthreads();
}