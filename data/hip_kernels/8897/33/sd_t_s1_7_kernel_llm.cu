#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sd_t_s1_7_kernel(int h1d,int h2d,int h3d,int p4d,int p6d,int p4ld_t2,int h1ld_t2,int h3ld_v2,int h2ld_v2,int p6ld_v2,int h3ld_t3,int h2ld_t3,int h1ld_t3,int p6ld_t3,int p4ld_t3,double *t3d, double *t2_d, double *v2_d,int p4, int total_x) {
    int h1, h2, h3, p6;
    __shared__ double t2_shm[T1 * 2 * Tcomm];

    // Load t2_d into shared memory with optimized loop boundary
    for(int i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        if(i < h1d * p4d)
            t2_shm[i] = t2_d[i];
    }
    __syncthreads();

    int base_idx = blockIdx.x * blockDim.x * T2 * T1;
    int thread_idx = base_idx + threadIdx.x;
    
    // Efficiently calculate indices within the loop
    for(int i = thread_idx; i < total_x; i += gridDim.x * blockDim.x) {
        int idx = i;
        h3 = idx % h3d;
        idx /= h3d;
        h2 = idx % h2d;
        idx /= h2d;
        p6 = idx % p6d;

        for(h1 = 0; h1 < h1d; h1++) {
            for(p4 = 0; p4 < p4d; p4++) {
                t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4 * p4ld_t3] += 
                    t2_shm[h1 * p4d + p4] * v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2];
            }
        }
    }
    __syncthreads();
}