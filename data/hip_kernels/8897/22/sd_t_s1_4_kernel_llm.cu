#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_4_kernel(size_t h1d, size_t h2d, size_t h3d, size_t p4d, size_t p5d, size_t p6d, size_t p4ld_t2, size_t h1ld_t2, size_t h3ld_v2, size_t h2ld_v2, size_t p6ld_v2, size_t p5ld_v2, size_t h3ld_t3, size_t h2ld_t3, size_t h1ld_t3, size_t p6ld_t3, size_t p5ld_t3, size_t p4ld_t3, double *t3d, double *t2_d, double *v2_d, size_t p4, size_t total_x) {
    size_t h1, h2, h3, p6, p5;
    __shared__ double t2_shm[T1 * 4 * Tcomm];

    // Optimize shared memory loading with stride loop and boundary check
    for (size_t i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        if (i < h1d * p4d)
            t2_shm[i] = t2_d[i];
    }
    
    size_t rest_x = blockIdx.x;
    size_t thread_x = T2 * T1 * rest_x + threadIdx.x;

    __syncthreads();
    
    // Modify loop to reduce redundancies
    for (size_t i = thread_x; i < total_x; i += gridDim.x * blockDim.x) {
        size_t idx = i;
        h3 = idx % h3d;
        idx /= h3d;
        h2 = idx % h2d;
        idx /= h2d;
        p6 = idx % p6d;
        idx /= p6d;
        p5 = idx % p5d;
        
        if (i < total_x) {
            for (h1 = 0; h1 < h1d; h1++) {
                for (p4 = 0; p4 < p4d; p4++) {
                    t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p5 * p5ld_t3 + p4 * p4ld_t3] -=
                        t2_shm[h1 * p4d + p4] * v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2 + p5 * p5ld_v2];
                }
            }
        }
    }
    __syncthreads();
}