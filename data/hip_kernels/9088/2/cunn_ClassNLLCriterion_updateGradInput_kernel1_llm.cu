#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

static const int NTHREADS = 32;

__global__ void cunn_ClassNLLCriterion_updateGradInput_kernel1(float* gradInput, float* weights, float* target, float* total_weight, int size_average, int n_classes)
{
    // Early exit if weighted total is zero or negative
    if (*total_weight <= 0) {
        return;
    }

    // Compute normalization factor
    float norm = size_average ? (1.0f / *total_weight) : 1.0f;

    // Compute thread ID for processing data
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index falls within valid range
    if (idx < n_classes) {
        int t = (int)target[idx] - 1; // Adjust for zero-based indexing
        if (t >= 0 && t < n_classes) {
            gradInput[t] = -(weights ? weights[t] : 1.0f) * norm; // Update gradient input
        }
    }
}
```
