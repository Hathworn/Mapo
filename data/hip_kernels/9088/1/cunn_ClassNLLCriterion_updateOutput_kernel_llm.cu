#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

static const int NTHREADS = 32;

__global__ void cunn_ClassNLLCriterion_updateOutput_kernel(float *output, float *total_weight, float *input, float *target, float *weights, int size_average, int nframe, int ndim, int n_classes) {
    __shared__ float shInputs[NTHREADS], acc_weight[NTHREADS];
    int i, t;
    float cur_weight;
    
    // Initialize shared memory
    shInputs[threadIdx.x] = 0.0f;
    acc_weight[threadIdx.x] = 0.0f;

    // Compute using strides over nframe
    for (i = threadIdx.x; i < nframe; i += blockDim.x) {
        t = target[i] - 1;
        if (t >= 0 && t < n_classes) {
            cur_weight = weights ? weights[t] : 1.0f;
            atomicAdd(&shInputs[threadIdx.x], -input[i * ndim + t] * cur_weight);
            atomicAdd(&acc_weight[threadIdx.x], cur_weight);
        }
    }
    __syncthreads();

    // Reduction and final update
    if (threadIdx.x == 0) {
        *output = *total_weight = 0;
        for (i = 0; i < NTHREADS; ++i){
            atomicAdd(output, shInputs[i]);
            atomicAdd(total_weight, acc_weight[i]);
        }
        if (size_average && *total_weight > 0) {
            *output /= *total_weight;
        }
    }
}