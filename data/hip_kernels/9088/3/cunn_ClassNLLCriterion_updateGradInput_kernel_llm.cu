#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

static const int NTHREADS = 32;

__global__ void cunn_ClassNLLCriterion_updateGradInput_kernel(float *gradInput, float *target, float *weights, float *total_weight, int size_average, int nframe, int ndim, int n_classes) 
{
    // Check total_weight once to minimize divergence
    if (*total_weight <= 0) return;

    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    float norm = size_average ? (1.0f / *total_weight) : 1.0f;

    // Use a single loop over all elements handled by this block
    while (i < nframe) {
        int t = (int)target[i] - 1; // Retrieve target class index
        if (t >= 0 && t < n_classes) {
            gradInput[i * ndim + t] = -(weights ? weights[t] : 1.0f) * norm;
        }
        i += blockDim.x * gridDim.x; // Increment by total number of threads
    }
}