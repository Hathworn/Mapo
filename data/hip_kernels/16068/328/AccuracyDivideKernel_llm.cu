#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load accuracy to register for faster access
    float localAccuracy = *accuracy;
    
    // Divide using register value
    localAccuracy /= N;
    
    // Store the result back to global memory
    *accuracy = localAccuracy;
}