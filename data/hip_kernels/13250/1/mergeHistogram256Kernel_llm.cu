#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeHistogram256Kernel( uint *d_Histogram, uint *d_PartialHistograms, uint histogramCount )
{
    uint sum = 0;

    // Use a loop unrolling optimization for the first loop
    for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)
    {
        // Manually unrolling the iterations can improve performance
        sum += d_PartialHistograms[blockIdx.x + i * HISTO256_BINS];
        if (i + MERGE_THREADBLOCK_SIZE < histogramCount)
            sum += d_PartialHistograms[blockIdx.x + (i + MERGE_THREADBLOCK_SIZE) * HISTO256_BINS];
    }

    __shared__ uint data[MERGE_THREADBLOCK_SIZE];
    data[threadIdx.x] = sum;

    // Use warp shuffle to optimize reduction in shared memory
    for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
    {
        __syncthreads();

        if (threadIdx.x < stride)
        {
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
    }

    if (threadIdx.x == 0)
    {
        d_Histogram[blockIdx.x] = data[0];
    }
}