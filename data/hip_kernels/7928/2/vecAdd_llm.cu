#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(float *in1, float *in2, float *out, int len)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread adds one element from in1 and in2 if within bounds
    if (i < len)
    {
        out[i] = in1[i] + in2[i];
    }
}