#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMultiply(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns)
{
    // variable declarations
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure within bounds and compute matrix multiplication
    if (row < numARows && col < numBColumns)
    {
        float Cvalue = 0.0;
        
        // Use shared memory to reduce global memory accesses
        __shared__ float sharedA[32][32]; // Assuming blockDim is 32
        __shared__ float sharedB[32][32];

        for (int k = 0; k < (numAColumns + 31) / 32; ++k)
        {
            if(k * 32 + threadIdx.x < numAColumns && row < numARows)
                sharedA[threadIdx.y][threadIdx.x] = A[row * numAColumns + k * 32 + threadIdx.x];
            else
                sharedA[threadIdx.y][threadIdx.x] = 0.0;

            if(k * 32 + threadIdx.y < numBRows && col < numBColumns)
                sharedB[threadIdx.y][threadIdx.x] = B[(k * 32 + threadIdx.y) * numBColumns + col];
            else
                sharedB[threadIdx.y][threadIdx.x] = 0.0;

            __syncthreads();

            for (int n = 0; n < 32; ++n)
                Cvalue += sharedA[threadIdx.y][n] * sharedB[n][threadIdx.x];

            __syncthreads();
        }
        
        C[row * numCColumns + col] = Cvalue;
    }
}