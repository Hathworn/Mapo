#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(float *in1, float *in2, float *out, int len)
{
    // Compute the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use conditional assignment to determine if the thread should perform the addition
    if (i < len)
    {
        out[i] = in1[i] + in2[i];
    }
}