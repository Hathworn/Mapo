#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

__global__ void vectorAdditionCUDA(const float* a, const float* b, float* c, int n)
{
    int ii = blockIdx.x * blockDim.x + threadIdx.x;
    if (ii < n)
    {
        // Perform vector addition
        c[ii] = a[ii] + b[ii];
    }
}