#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulKernel(float *Md, float *Nd, float *Pd, int Width) {
    // Use shared memory to reduce global memory accesses
    __shared__ float Mds[16][16]; // Assume 16x16 block size for illustration
    __shared__ float Nds[16][16];

    // 2D Thread ID
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = blockIdx.y * blockDim.y + ty;
    int Col = blockIdx.x * blockDim.x + tx;

    float Pvalue = 0;

    // Iterate over tiles
    for (int t = 0; t < (Width + 15) / 16; ++t) {  // Assuming block width of 16
        // Load data into shared memory
        if (Row < Width && t * 16 + tx < Width) {
            Mds[ty][tx] = Md[Row * Width + t * 16 + tx];
        }
        if (Col < Width && t * 16 + ty < Width) {
            Nds[ty][tx] = Nd[(t * 16 + ty) * Width + Col];
        }

        __syncthreads(); // Synchronize to ensure tiles are loaded

        // Compute product for the current tile
        for (int k = 0; k < 16; ++k) {
            Pvalue += Mds[ty][k] * Nds[k][tx];
        }

        __syncthreads(); // Synchronize to ensure completion of calculation for this tile
    }

    // Write the result to global memory
    if (Row < Width && Col < Width) {
        Pd[Row * Width + Col] = Pvalue;
    }
}