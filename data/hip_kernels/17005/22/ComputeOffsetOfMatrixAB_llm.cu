#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixAB(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, int32_t K_A_B, int32_t N) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within bounds
    if (i < N) {
        output[blockIdx.x * N + i] = K_A_B - row_sum[blockIdx.x] - col_sum[i];
    }
}