#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PadMatrixInLeadingDimensionKernel(const int8_t* src, int8_t* dst, int col_src, int col_dst) {
    // Load block index and shared memory padding computation
    int row_offset_src = blockIdx.x * col_src;
    int row_offset_dst = blockIdx.x * col_dst;
    
    for (int32_t i = threadIdx.x; i < col_src; i += blockDim.x) {
        // Improved memory access with precomputed offsets
        dst[row_offset_dst + i] = src[row_offset_src + i];
    }
}