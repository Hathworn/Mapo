#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    int32_t idx = blockIdx.x * N + threadIdx.x;  // Calculate the index once
    int32_t stride = blockDim.x;  // Define the stride
    
    // Iterate through indices with the calculated stride
    for (int32_t i = threadIdx.x; i < N; i += stride) {
        output[idx + i] = -row_sum[blockIdx.x];  // Directly modify the target element
    }
}