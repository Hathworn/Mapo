#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Use shared memory for better performance
    extern __shared__ int64_t shared[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < num_elements) {
        to_normalize[idx * 3] = batch_index; // Efficient memory coalescing
        to_normalize[idx * 3 + 1] = class_index;
        to_normalize[idx * 3 + 2] = static_cast<int64_t>(original[idx]);
    }
}