#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    // Use blockDim.x * blockIdx.x to compute global thread index
    int32_t globalIndex = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Iterate over elements with globalIndex and stride of total threads in the grid
    int32_t stride = gridDim.x * blockDim.x;
    for (int32_t i = globalIndex; i < N; i += stride) {
        output[blockIdx.x * N + i] = -col_sum[i];
    }
}