#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelA(float* r, float* x, float* y, float* z, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int stride = blockDim.x * gridDim.x; // Calculate stride
    for (int i = index; i < size; i += stride) {
        r[i] = x[i] * y[i] + z[i];
    }
}