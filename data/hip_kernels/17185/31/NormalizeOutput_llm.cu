#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop through elements with stride to improve parallelism
    while (idx < num_elements) {
        // Unroll the loop by manually setting each index of to_normalize
        int64_t offset = idx * 3;
        to_normalize[offset] = batch_index;
        to_normalize[offset + 1] = class_index;
        to_normalize[offset + 2] = static_cast<int64_t>(original[idx]);
        
        // Move to the next element
        idx += blockDim.x * gridDim.x;
    }
}