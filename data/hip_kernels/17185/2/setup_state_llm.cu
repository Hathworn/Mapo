#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique thread ID.
    hiprand_init(seed, id, 0, &state[id]); // Use thread ID to initialize state array.
}