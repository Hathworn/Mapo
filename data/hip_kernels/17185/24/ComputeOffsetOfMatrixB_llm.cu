#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        // Optimize access by removing multiplication from the loop
        int offset = -row_sum[blockIdx.x];
        
        for (int i = idx; i < N; i += blockDim.x) {
            *(output + blockIdx.x * N + i) = offset;
        }
    }
}