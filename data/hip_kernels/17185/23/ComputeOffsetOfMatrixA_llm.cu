#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    // Calculate global thread index
    int32_t tid = blockIdx.x * blockDim.x + threadIdx.x; 
    // Loop through matrix elements with step size of total threads
    for (int32_t i = tid; i < N; i += gridDim.x * blockDim.x) { 
        *(output + blockIdx.x * N + i) = -col_sum[i];
    }
}