#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define TOLERANCE 0.00001
#define TRUE 1
#define FALSE 0

long usecs();
void initialize(double **A, int rows, int cols);
int calc_serial(double **A, int rows, int cols, int iters, double tolerance);
int calc_serial_v1(double **A, int rows, int cols, int iters, double tolerance);
int calc_omp(double **A, int rows, int cols, int iters, double tolerance, int num_threads);
int calc_gpu(double **A, int rows, int cols, int iters, double tolerance);
double verify(double **A, double **B, int rows, int cols);


__global__ void calc_kernel(double* w, double* r, int rows, int cols, double tolerance) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure row and column indices are within bounds
    if (row >= 1 && row < rows - 1 && col >= 1 && col < cols - 1) {
        int idx = row * cols + col; // Correct row-major order index calculation
        // Perform calculation with boundary checks
        w[idx] = 0.2 * (r[idx + 1] + r[idx - 1] + r[(row - 1) * cols + col] + r[(row + 1) * cols + col]); 
    }
}