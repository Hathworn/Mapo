#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
const float PI = 3.14159265359;
const float HALFPI = 0.5*PI;
texture<unsigned char, hipTextureType3D, hipReadModeElementType> tcExpData;
texture<float, hipTextureType2D, hipReadModeElementType> tfG;

typedef struct {
    int iNPixelJ, iNPixelK;
    float fPixelJ, fPixelK;
    float afCoordOrigin[3];
    float afNorm[3];
    float afJVector[3];
    float afKVector[3];
    float fNRot, fAngleStart, fAngleEnd;
} DetInfo;

__device__ void d_euler_zxz_to_mat(float* afEuler, float* afMat) {
    float s1 = sin(afEuler[0]);
    float s2 = sin(afEuler[1]);
    float s3 = sin(afEuler[2]);
    float c1 = cos(afEuler[0]);
    float c2 = cos(afEuler[1]);
    float c3 = cos(afEuler[2]);
    afMat[0] = c1 * c3 - c2 * s1 * s3;
    afMat[1] = -c1 * s3 - c3 * c2 * s1;
    afMat[2] = s1 * s2;
    afMat[3] = s1 * c3 + c2 * c1 * s3;
    afMat[4] = c1 * c2 * c3 - s1 * s3;
    afMat[5] = -c1 * s2;
    afMat[6] = s3 * s2;
    afMat[7] = s2 * c3;
    afMat[8] = c2;
}

__global__ void rand_mat_neighb_from_euler(float* afEulerIn, float* afMatOut, float* afRand, float fBound) {
    // Reduce redundant calculations by precomputing indices and offset base
    int eulerIndex = threadIdx.x * 3;
    int randIndexBase = blockIdx.x * blockDim.x * 3;
    int matIndexBase = blockIdx.x * blockDim.x * 9;
    
    // Compute afEulerTmp using cached indices
    float afEulerTmp[3];
    afEulerTmp[0] = afEulerIn[eulerIndex + 0] + (2 * afRand[randIndexBase + eulerIndex + 0] - 1) * fBound;
    afEulerTmp[2] = afEulerIn[eulerIndex + 2] + (2 * afRand[randIndexBase + eulerIndex + 2] - 1) * fBound;
    float z = cos(afEulerIn[eulerIndex + 1]) +
              (afRand[randIndexBase + eulerIndex + 1] * 2 - 1) * sin(afEulerIn[eulerIndex + 1] * fBound);
    
    // Use clamp to restrict the value of z
    afEulerTmp[1] = acosf(fminf(fmaxf(z, -1.0f), 1.0f));

    // Write output matrix using cached indices
    int matOutIndex = matIndexBase + threadIdx.x * 9;
    if (blockIdx.x > 0) {
        d_euler_zxz_to_mat(afEulerTmp, afMatOut + matOutIndex);
    } else {
        d_euler_zxz_to_mat(afEulerIn + eulerIndex, afMatOut + matOutIndex);
    }
}