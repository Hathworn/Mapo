#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const float PI = 3.14159265359;
const float HALFPI = 0.5*PI;
texture<unsigned char, hipTextureType3D, hipReadModeElementType> tcExpData;
texture<float, hipTextureType2D, hipReadModeElementType> tfG;

typedef struct {
    int iNPixelJ, iNPixelK;
    float fPixelJ, fPixelK;
    float afCoordOrigin[3];
    float afNorm[3];
    float afJVector[3];
    float afKVector[3];
    float fNRot, fAngleStart, fAngleEnd;
} DetInfo;

__device__ void mat3_dot(float* afResult, float* afM0, float* afM1) {
    // Optimized dot product of two 3x3 matrices
    for (int i = 0; i < 3; i++) {
        afResult[i * 3]     = afM0[i * 3] * afM1[0] + afM0[i * 3 + 1] * afM1[3] + afM0[i * 3 + 2] * afM1[6];
        afResult[i * 3 + 1] = afM0[i * 3] * afM1[1] + afM0[i * 3 + 1] * afM1[4] + afM0[i * 3 + 2] * afM1[7];
        afResult[i * 3 + 2] = afM0[i * 3] * afM1[2] + afM0[i * 3 + 1] * afM1[5] + afM0[i * 3 + 2] * afM1[8];
    }
}

__device__ void mat3_transpose(float* afOut, float* afIn) {
    // Optimized transpose of 3x3 matrix
    afOut[0] = afIn[0]; afOut[1] = afIn[3]; afOut[2] = afIn[6];
    afOut[3] = afIn[1]; afOut[4] = afIn[4]; afOut[5] = afIn[7];
    afOut[6] = afIn[2]; afOut[7] = afIn[5]; afOut[8] = afIn[8];
}

__global__ void misorien(float* afMisOrien, float* afM0, float* afM1, float* afSymM) {
    // Optimized kernel function for misorientation calculation
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    float afTmp0[9];
    float afTmp1[9];
    float afM1Transpose[9];
    float fCosAngle;
    
    mat3_transpose(afM1Transpose, &afM1[blockIdx.x * 9]);
    mat3_dot(afTmp0, &afSymM[threadIdx.x * 9], afM1Transpose);
    mat3_dot(afTmp1, &afM0[blockIdx.x * 9], afTmp0);
    
    fCosAngle = 0.5f * (afTmp1[0] + afTmp1[4] + afTmp1[8] - 1);
    fCosAngle = min(0.9999999999f, fCosAngle);
    fCosAngle = max(-0.99999999999f, fCosAngle);
    
    afMisOrien[i] = acosf(fCosAngle);
}