#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const float PI = 3.14159265359;
const float HALFPI = 0.5*PI;
texture<unsigned char, hipTextureType3D, hipReadModeElementType> tcExpData;
texture<float, hipTextureType2D, hipReadModeElementType> tfG;  // texture to store scattering vectors;
typedef struct {
    int iNPixelJ, iNPixelK;
    float fPixelJ, fPixelK;
    float afCoordOrigin[3];
    float afNorm[3];
    float afJVector[3];
    float afKVector[3];
    float fNRot, fAngleStart, fAngleEnd;
} DetInfo;

__global__ void mat_to_euler_ZXZ(float* afMatIn, float* afEulerOut, int iNAngle) {
    /*
    * transform active rotation matrix to euler angles in ZXZ convention, not right(seems right now)
    * afMatIn: iNAngle * 9
    * afEulerOut: iNAngle* 3
    * TEST PASSED
    */
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float threshold = 0.9999999;
    
    if (i < iNAngle) {
        float m22 = afMatIn[i * 9 + 8];
        if (m22 > threshold) {
            // Handle special case when m[2, 2] is near 1
            afEulerOut[i * 3 + 0] = 0;
            afEulerOut[i * 3 + 1] = 0;
            afEulerOut[i * 3 + 2] = atan2(afMatIn[i * 9 + 3], afMatIn[i * 9]); 
        } else if (m22 < -threshold) {
            // Handle special case when m[2, 2] is near -1
            afEulerOut[i * 3 + 0] = 0;
            afEulerOut[i * 3 + 1] = PI;
            afEulerOut[i * 3 + 2] = atan2(afMatIn[i * 9 + 1], afMatIn[i * 9]);
        } else {
            // General case
            float m20 = afMatIn[i * 9 + 6];
            float m21 = afMatIn[i * 9 + 7];
            float m12 = afMatIn[i * 9 + 5];
            float atan20 = atan2(m20, m21); 

            afEulerOut[i * 3 + 0] = atan2(afMatIn[i * 9 + 2], -m12);
            afEulerOut[i * 3 + 1] = atan2(sqrt(m20 * m20 + m21 * m21), m22);
            afEulerOut[i * 3 + 2] = atan20;

            // Ensure all angles are positive
            if (afEulerOut[i * 3 + 0] < 0) afEulerOut[i * 3 + 0] += 2 * PI;
            if (afEulerOut[i * 3 + 1] < 0) afEulerOut[i * 3 + 1] += 2 * PI;
            if (afEulerOut[i * 3 + 2] < 0) afEulerOut[i * 3 + 2] += 2 * PI;
        }
    }
}