#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
const float PI = 3.14159265359;
const float HALFPI = 0.5*PI;
texture<unsigned char, hipTextureType3D, hipReadModeElementType> tcExpData;
texture<float, hipTextureType2D, hipReadModeElementType> tfG;  // texture to store scattering vectors;

typedef struct {
    int iNPixelJ, iNPixelK;
    float fPixelJ, fPixelK;
    float afCoordOrigin[3];
    float afNorm[3];
    float afJVector[3];
    float afKVector[3];
    float fNRot, fAngleStart, fAngleEnd;
} DetInfo;

// Global kernel function to display random numbers
__global__ void display_rand(float* afRandom, int iNRand) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds to prevent out-of-range access
    if (i < iNRand) {
        printf("=%d= %f ||", i, afRandom[i]);
    }
}