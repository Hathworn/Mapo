#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const float PI = 3.14159265359;
const float HALFPI = 0.5*PI;
texture<unsigned char, hipTextureType3D, hipReadModeElementType> tcExpData;
texture<float, hipTextureType2D, hipReadModeElementType> tfG;  // texture to store scattering vectors;
typedef struct {
    int iNPixelJ, iNPixelK;
    float fPixelJ, fPixelK;
    float afCoordOrigin[3];
    float afNorm[3];
    float afJVector[3];
    float afKVector[3];
    float fNRot, fAngleStart, fAngleEnd;
} DetInfo;

__global__ void euler_zxz_to_mat(float* afEuler, float* afMat, int iNAngle) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Only proceed if thread index is within bounds
    if (i < iNAngle) {
        float a = afEuler[i * 3 + 0];
        float b = afEuler[i * 3 + 1];
        float c = afEuler[i * 3 + 2];

        float s1 = sin(a);
        float s2 = sin(b);
        float s3 = sin(c);
        float c1 = cos(a);
        float c2 = cos(b);
        float c3 = cos(c);

        // Efficiently calculate matrix elements
        afMat[i * 9 + 0] = c1 * c3 - c2 * s1 * s3;
        afMat[i * 9 + 1] = -c1 * s3 - c3 * c2 * s1;
        afMat[i * 9 + 2] = s1 * s2;
        afMat[i * 9 + 3] = s1 * c3 + c2 * c1 * s3;
        afMat[i * 9 + 4] = c1 * c2 * c3 - s1 * s3;
        afMat[i * 9 + 5] = -c1 * s2;
        afMat[i * 9 + 6] = s3 * s2;
        afMat[i * 9 + 7] = s2 * c3;
        afMat[i * 9 + 8] = c2;
    }
}