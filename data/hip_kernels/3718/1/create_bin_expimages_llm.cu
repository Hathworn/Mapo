#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const float PI = 3.14159265359;
const float HALFPI = 0.5*PI;
texture<unsigned char, hipTextureType3D, hipReadModeElementType> tcExpData;
texture<float, hipTextureType2D, hipReadModeElementType> tfG;

typedef struct {
    int iNPixelJ, iNPixelK;
    float fPixelJ, fPixelK;
    float afCoordOrigin[3];
    float afNorm[3];
    float afJVector[3];
    float afKVector[3];
    float fNRot, fAngleStart, fAngleEnd;
} DetInfo;

__global__ void create_bin_expimages(char* acExpDetImages, const int* aiDetStartIdx, const float* afDetInfo, const int iNDet, const int iNRot, const int* aiDetIndex, const int* aiRotN, const int* aiJExp, const int* aiKExp, const int iNPeak) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread operates within bounds
    if (i < iNPeak) {
        // Pre-compute indices
        int detIndex = aiDetIndex[i];
        int startIdx = aiDetStartIdx[detIndex];
        int nPixelJ = int(afDetInfo[0 + 19 * detIndex]);
        int nPixelK = int(afDetInfo[1 + 19 * detIndex]);
        
        // Calculate the offset for image indexing
        int offset = startIdx + aiRotN[i] * nPixelJ * nPixelK + aiKExp[i] * nPixelJ + aiJExp[i];
        
        // Set the image pixel to 1
        acExpDetImages[offset] = 1;
    }
}