#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define block_count 32
#define thread_per_block 1024

// Optimized kernel
__global__ void GEMMLowpKernel(const float* __restrict__ in, const int N, float* __restrict__ out, float scale, float shift, long long qmax, const float* __restrict__ noise, bool enforce_true_zero) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure thread operates within bounds
    if (i < N) {
        // Load input with noise
        float temp_out = in[i] + noise[i];
        // Conditionally scale and shift
        if (enforce_true_zero) {
            temp_out = (temp_out / scale) + shift;
        } else {
            temp_out = (temp_out + shift) / scale;
        }
        // Clip and round value
        temp_out = fminf(fmaxf(roundf(temp_out), 0.0f), static_cast<float>(qmax));
        // Conditionally revert scaling and shifting
        if (enforce_true_zero) {
            out[i] = (temp_out - shift) * scale;
        } else {
            out[i] = temp_out * scale - shift;
        }
    }
}