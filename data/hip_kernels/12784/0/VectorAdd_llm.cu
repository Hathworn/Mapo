#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 100
#define GRID_SIZE 100
#define N GRID_SIZE * BLOCK_SIZE

__global__ void VectorAdd(int *A, int *B, int *C) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < N) { // Ensure within bounds
        C[x] = A[x] + B[x];
    }
}