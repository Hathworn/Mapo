#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void printHello() {
    // Use blockIdx and threadIdx to ensure output from only one thread
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        printf("The Device says \"Hello World\"\n");
    }
}