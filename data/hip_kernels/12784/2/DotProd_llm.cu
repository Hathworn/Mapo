#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DotProd(int *a, int *b, int *c) {
    __shared__ int temp[THREADS_PER_BLOCK];

    int x = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Calculate directly without printing debug information
    temp[threadIdx.x] = a[x] * b[x];

    __syncthreads();

    if (threadIdx.x == 0) {
        int sum = 0;
        // Use a loop unrolling technique for performance improvement
        #pragma unroll
        for (int i = 0; i < THREADS_PER_BLOCK; i++) {
            sum += temp[i];
        }

        atomicAdd(c, sum);
    }
}