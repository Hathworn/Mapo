#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory to reduce kernel launch overhead
    __shared__ float shared_accuracy;
    
    // Only one thread in the block performs the division
    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy / N;
    }
    
    // Ensure all threads have the updated value
    __syncthreads();
    
    // Copy the result back to global memory (only needed once)
    if (threadIdx.x == 0) {
        *accuracy = shared_accuracy;
    }
}