#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    unsigned int* orig_ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* ptr = orig_ptr;
    
    // Return early if original pointer exceeds end pointer
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);
    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Initial memory write
    *ptr = pattern;

    // Loop to write patterns within block boundary
    while (ptr < block_end)
    {
        ptr = (unsigned int*)((unsigned long)orig_ptr | mask);

        // Check if mask produces original pointer, adjust mask
        if (ptr == orig_ptr) {
            mask = mask << 1;
            continue;
        }

        // If calculated pointer exceeds block end, exit loop
        if (ptr >= block_end) {
            break;
        }

        // Write pattern to calculated pointer
        *ptr = pattern;

        pattern = pattern << 1;
        mask = mask << 1;
    }
    return;
}