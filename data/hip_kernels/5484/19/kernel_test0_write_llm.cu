#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);

    // Early exit if block overflows
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Use unrolled loops for faster memory writes
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        ptr[i] = pattern;
    }

    return;
}