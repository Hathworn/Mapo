#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_init(char* _ptr, char* end_ptr)
{
    // Calculate initial pointer offset based on block index
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Early exit if the pointer exceeds end_ptr
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int p1 = 1;
    unsigned int p2;
    unsigned int step = BLOCKSIZE / sizeof(unsigned int);

    #pragma unroll // Unroll loop to improve performance
    for (unsigned int i = 0; i < step; i += 16){
        p2 = ~p1; // Calculate complement of p1

        // Assign values to memory in one block
        ptr[i] = ptr[i+1] = ptr[i+4] = ptr[i+5] = ptr[i+8] = ptr[i+9] = ptr[i+12] = ptr[i+13] = p1;
        ptr[i+2] = ptr[i+3] = ptr[i+6] = ptr[i+7] = ptr[i+10] = ptr[i+11] = ptr[i+14] = ptr[i+15] = p2;

        p1 <<= 1; // Shift p1 left by 1
        if (p1 == 0) {
            p1 = 1; // Reset p1 if it becomes 0
        }
    }
}