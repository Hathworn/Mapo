#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    // Get starting index for this thread/block
    unsigned int idx = blockIdx.x * BLOCKSIZE + threadIdx.x * sizeof(unsigned int); 
    unsigned int* ptr = (unsigned int*) (_ptr + idx);
    unsigned int* start_ptr = (unsigned int*) _start_ptr;

    // Return early if out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Calculate number of iterations per thread
    unsigned int iterations = BLOCKSIZE / sizeof(unsigned int) / blockDim.x;

    // Use shared memory to optimize
    __shared__ unsigned int shared_data[BLOCKSIZE / sizeof(unsigned int)];

    // Load data into shared memory
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        shared_data[i] = start_ptr[i];
    }
    __syncthreads();

    // Write from shared memory to global memory
    for (unsigned int j = 0; j < iterations; j++) {
        if (idx / sizeof(unsigned int) + j * blockDim.x < BLOCKSIZE / sizeof(unsigned int)) {
            ptr[j * blockDim.x] = shared_data[idx / sizeof(unsigned int) + j * blockDim.x];
        }
    }
}