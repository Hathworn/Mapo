#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    unsigned int tid = threadIdx.x; // Utilize thread index for parallel processing
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);

    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int half_count = BLOCKSIZE / sizeof(unsigned int) / 2;
    unsigned int* ptr_mid = ptr + half_count;

    // Parallelize operations using threadIdx
    if (tid < half_count) {
        ptr_mid[tid] = ptr[tid];
    }
    __syncthreads(); // Ensure ptr_mid is completely written

    if (tid < half_count - 8) {
        ptr[tid + 8] = ptr_mid[tid];
    }
    __syncthreads(); // Ensure first operation phase is complete

    if (tid < 8) {
        ptr[tid] = ptr_mid[half_count - 8 + tid];
    }
}