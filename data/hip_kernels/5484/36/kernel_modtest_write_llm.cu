#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    // Pre-calculate useful constants for loop conditions
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int total_elements = BLOCKSIZE / sizeof(unsigned int);
    
    // Early exit if pointer exceeds the end address
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Use single loop with conditional for write efficiency
    for (unsigned int i = 0; i < total_elements; i++) {
        if (i % MOD_SZ == offset) {
            ptr[i] = p1;  // Write p1 at positions matching offset
        } else {
            ptr[i] = p2;  // Otherwise write p2
        }
    }

    return;
}