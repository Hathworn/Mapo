#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int* orig_ptr = ptr;

    unsigned int pattern = 1;
    unsigned long mask = 4;

    *ptr = pattern;

    // Use a more efficient loop control with fewer checks
    while (true) {
        ptr = (unsigned int*)(((unsigned long)orig_ptr) | mask);
        
        if (ptr >= end_ptr) {
            break; // Exit if pointer exceeds boundary
        }

        *ptr = pattern; // Write pattern to memory

        pattern <<= 1;  // Increment pattern by shifting left
        mask <<= 1;     // Shift mask to next position
    }
}