#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Ensure that the pointer calculation is within valid range
    if (ptr >= (unsigned long*) end_ptr) {
        return;
    }

    // Use threadIdx.x for parallel writes if possible
    int idx = threadIdx.x;
    for (int i = idx; i < BLOCKSIZE / sizeof(unsigned long); i += blockDim.x){
        ptr[i] = (unsigned long) &ptr[i];
    }

    return;
}