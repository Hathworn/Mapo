#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset) {
    // Calculate the initial pointer for the current block
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);

    // Terminate if the pointer is beyond the end
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Calculate thread-local indices and stride
    unsigned int tid = threadIdx.x;
    unsigned int stride = blockDim.x;

    // Initialize pattern variables
    unsigned int k = offset;
    unsigned pat = pattern;

    // Loop through with strided access for threading
    for (unsigned int i = tid; i < BLOCKSIZE / sizeof(unsigned int); i += stride) {
        ptr[i] = pat;
        k++;
        if (k >= 32) {
            k = 0;
            pat = lb;
        } else {
            pat = (pat << 1) | sval;
        }
    }
}