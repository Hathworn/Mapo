#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    const uint totalElements = tgtWidth * tgtHeight;

    // Use shared memory to reduce global memory access
    __shared__ float tile[blockDim.x];

    for (uint i = idx; i < totalElements; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        
        // Prefetch data to shared memory
        tile[threadIdx.x] = src[srcY * srcWidth + srcX];
        __syncthreads();

        // Write the prefetched data to global memory
        tgt[i] = tile[threadIdx.x];
    }
}