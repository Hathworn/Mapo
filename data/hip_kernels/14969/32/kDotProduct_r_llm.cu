#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numCols, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0;  // Accumulate in a register for faster access
    if (eidx < numCols) {
        for (; eidx < numElements; eidx += numCols) {
            sum += a[eidx] * b[eidx];
        }
    }
    shmem[threadIdx.x] = sum;  // Store result in shared memory
    __syncthreads();

    // Reduce on shared memory using a loop to handle cases where DP_BLOCKSIZE may not be a power of 2
    for (uint stride = DP_BLOCKSIZE / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];  // Write block result to global memory
    }
}