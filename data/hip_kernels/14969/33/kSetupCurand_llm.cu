#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize by adjusting thread block size and using efficient indexing
__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use more efficient thread index calculation
    hiprand_init(seed, idx, 0, &state[idx]);
}