#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrollWarp8(int *g_idata, int *g_odata, unsigned int n)
{
    // Set the thread id.
    unsigned int tid = threadIdx.x;
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x * 8;
    
    // Convert global data pointer to the local pointer of this block.
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // Unrolling 8 data blocks.
    int sum = 0;
    if (idx + blockDim.x * 7 < n)
    {
        sum += g_idata[idx];
        sum += g_idata[idx + blockDim.x];
        sum += g_idata[idx + blockDim.x * 2];
        sum += g_idata[idx + blockDim.x * 3];
        sum += g_idata[idx + blockDim.x * 4];
        sum += g_idata[idx + blockDim.x * 5];
        sum += g_idata[idx + blockDim.x * 6];
        sum += g_idata[idx + blockDim.x * 7];
    }
    idata[tid] = sum;
    __syncthreads();

    // In-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        // Synchronize within threadblock.
        __syncthreads();
    }

    // Unrolling warp
    if (tid < 32)
    {
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    // Write result for this block to global mem.
    if (tid == 0)
    {
        g_odata[blockIdx.x] = idata[0];
    }
}