#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll4Row(int *in, int *out, const int nx, const int ny)
{
    // Calculate thread id with consideration of warp granularity
    unsigned int ix = (threadIdx.x + blockIdx.x * blockDim.x) * 4;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix < nx - 3 * blockDim.x && iy < ny) // Check with optimized boundary condition
    {
        unsigned int ti = iy * nx + ix; // Access in rows
        unsigned int to = ix * ny + iy; // Access in cols

        // Unrolling loads for improved coalescing
        int temp0 = in[ti];
        int temp1 = in[ti + blockDim.x];
        int temp2 = in[ti + blockDim.x * 2];
        int temp3 = in[ti + blockDim.x * 3];

        // Unrolling stores
        out[to] = temp0;
        out[to + ny * blockDim.x] = temp1;
        out[to + ny * blockDim.x * 2] = temp2;
        out[to + ny * blockDim.x * 3] = temp3;
    }
}