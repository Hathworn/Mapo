#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Use shared memory for MatC to reduce global memory accesses
    extern __shared__ float sMatC[];

    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny) {
        // Load data into shared memory
        sMatC[threadIdx.y * blockDim.x + threadIdx.x] = MatA[idx] + MatB[idx];
        __syncthreads();

        // Store result back to global memory
        MatC[idx] = sMatC[threadIdx.y * blockDim.x + threadIdx.x];
    }
}