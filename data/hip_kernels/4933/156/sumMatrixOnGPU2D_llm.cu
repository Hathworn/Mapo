#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Get global thread index
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Cache computation outside of condition
    unsigned int idx = iy * nx + ix;
    
    // Check within bounds
    if (ix < nx && iy < ny) {
        // Perform addition
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}