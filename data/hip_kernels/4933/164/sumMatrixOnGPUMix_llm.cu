#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate row and column for the current thread
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure thread index does not exceed matrix dimensions
    if (ix < nx && iy < ny) {
        unsigned int idx = iy * nx + ix; // Calculate global index
        MatC[idx] = MatA[idx] + MatB[idx]; // Element-wise addition of matrices
    }
}