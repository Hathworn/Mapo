#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate global thread index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Pre-calculate offset position
    unsigned int k = i + offset;

    // Use shared memory for improved performance (if available)
    extern __shared__ float sharedDataA[];
    extern __shared__ float sharedDataB[];

    // Load data into shared memory (optional optimization)
    if (k < n) {
        sharedDataA[threadIdx.x] = A[k];
        sharedDataB[threadIdx.x] = B[k];
        __syncthreads();

        // Ensure memory boundaries are respected
        C[i] = sharedDataA[threadIdx.x] + sharedDataB[threadIdx.x];
    }
}