#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyRow(int *in, int *out, const int nx, const int ny)
{
    // Compute global thread id
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idy = threadIdx.y + blockIdx.y * blockDim.y;

    // Flatten 2D grid into linear index
    unsigned int index = idy * nx + idx;

    // Ensure index is within array bounds
    if (idx < nx && idy < ny)
    {
        // Perform copy operation
        out[index] = in[index];
    }
}