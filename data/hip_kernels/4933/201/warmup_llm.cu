#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    // Use shared memory to minimize global memory accesses
    extern __shared__ float sharedMem[];
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Ensure thread index is within bounds
    if (i < n && k < n) {
        sharedMem[threadIdx.x] = A[k] + B[k]; // Load data to shared memory
        __syncthreads(); // Sync threads to ensure completion of shared memory loads

        C[i] = sharedMem[threadIdx.x]; // Write result back to global memory
    }
}