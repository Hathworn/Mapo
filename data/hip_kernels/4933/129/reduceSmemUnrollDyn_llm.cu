#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmemUnrollDyn(int *g_idata, int *g_odata, unsigned int n)
{
    extern __shared__ int smem[];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // unrolling 4
    int tmpSum = 0;

    if (idx + 3 * blockDim.x < n)
    {
        tmpSum = g_idata[idx] + g_idata[idx + blockDim.x] + 
                 g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x];
    }

    smem[tid] = tmpSum;
    __syncthreads();

    // perform reduction on shared memory
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) smem[tid] += smem[tid + s];
        __syncthreads();
    }

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = smem;  // volatile to prevent compiler reordering
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}