#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iKernel(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop for better performance, assuming N is a multiple of 4
    int step = 4;
    for (; i < N; i += blockDim.x * gridDim.x * step) {
        if (i < N) C[i] = A[i] + B[i];
        if (i + 1 < N) C[i + 1] = A[i + 1] + B[i + 1];
        if (i + 2 < N) C[i + 2] = A[i + 2] + B[i + 2];
        if (i + 3 < N) C[i + 3] = A[i + 3] + B[i + 3];
    }
}