#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeDiagonalColUnroll4(float *out, float *in, const int nx, const int ny)
{
    // Calculate optimized block coordinates
    unsigned int blk_y = blockIdx.x;
    unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

    // Calculate global thread indices with unrolling factor
    unsigned int ix_stride = blockDim.x * blk_x * 4;
    unsigned int ix = ix_stride + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    // Unroll 4 - Load and store
    if (ix < nx && iy < ny) {
        unsigned int out_index = iy * nx + ix;
        unsigned int in_index = ix * ny + iy;
    
        out[out_index] = in[in_index];
        if (ix + blockDim.x < nx) {
            out[out_index + blockDim.x] = in[in_index + blockDim.x * ny];
        }
        if (ix + 2 * blockDim.x < nx) {
            out[out_index + 2 * blockDim.x] = in[in_index + 2 * blockDim.x * ny];
        }
        if (ix + 3 * blockDim.x < nx) {
            out[out_index + 3 * blockDim.x] = in[in_index + 3 * blockDim.x * ny];
        }
    }
}