#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling8New(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // Unrolling 8 with bounds checking
    int tmp = 0;
    if (idx + 7 * blockDim.x < n) {
        int *ptr = g_idata + idx;

        // Increment tmp 8 times with values strided by blockDim.x
        #pragma unroll // Unroll the loop for performance
        for (int i = 0; i < 8; i++) {
            tmp += *ptr;
            ptr += blockDim.x;
        }

        idata[tid] = tmp; // Store result in shared memory
    } else if (idx < n) {
        idata[tid] = g_idata[idx]; // Handle remaining elements
    } else {
        idata[tid] = 0; // Avoid uninitialized memory
    }

    __syncthreads();

    // In-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}