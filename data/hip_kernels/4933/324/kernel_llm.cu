#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *g_data, float value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Coalesced memory access for improved memory bandwidth usage
    if (idx < gridDim.x * blockDim.x) {
        g_data[idx] += value;
    }
}