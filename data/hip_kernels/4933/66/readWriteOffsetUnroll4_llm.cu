#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readWriteOffsetUnroll4(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    unsigned int k = i + offset;
    
    // Use shared memory for faster access
    __shared__ float sharedA[1024];
    __shared__ float sharedB[1024];

    // Preload data to shared memory if within bounds
    if (k < n) {
        sharedA[threadIdx.x] = A[k];
        sharedB[threadIdx.x] = B[k];
    }
    if (k + blockDim.x < n) {
        sharedA[threadIdx.x + blockDim.x] = A[k + blockDim.x];
        sharedB[threadIdx.x + blockDim.x] = B[k + blockDim.x];
    }
    if (k + 2 * blockDim.x < n) {
        sharedA[threadIdx.x + 2 * blockDim.x] = A[k + 2 * blockDim.x];
        sharedB[threadIdx.x + 2 * blockDim.x] = B[k + 2 * blockDim.x];
    }
    if (k + 3 * blockDim.x < n) {
        sharedA[threadIdx.x + 3 * blockDim.x] = A[k + 3 * blockDim.x];
        sharedB[threadIdx.x + 3 * blockDim.x] = B[k + 3 * blockDim.x];
    }

    // Synchronize threads after loading to shared memory
    __syncthreads();

    // Conditional memory operations
    if (k + 3 * blockDim.x < n) {
        C[k]                  = sharedA[threadIdx.x]                  + sharedB[threadIdx.x];
        C[k + blockDim.x]     = sharedA[threadIdx.x + blockDim.x]     + sharedB[threadIdx.x + blockDim.x];
        C[k + 2 * blockDim.x] = sharedA[threadIdx.x + 2 * blockDim.x] + sharedB[threadIdx.x + 2 * blockDim.x];
        C[k + 3 * blockDim.x] = sharedA[threadIdx.x + 3 * blockDim.x] + sharedB[threadIdx.x + 3 * blockDim.x];
    }
}