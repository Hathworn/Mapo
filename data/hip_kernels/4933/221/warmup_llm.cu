#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n && i < n) // Added check to ensure i is within bounds
    {
        C[k] = A[i] + B[i]; // Perform computation only if both indices are valid
    }
}