#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceCompleteUnrollWarps8 (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8 with boundary check
    if (idx + 7 * blockDim.x < n)
    {
        int sum = g_idata[idx] + g_idata[idx + blockDim.x] + g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x]
                + g_idata[idx + 4 * blockDim.x] + g_idata[idx + 5 * blockDim.x] + g_idata[idx + 6 * blockDim.x] + g_idata[idx + 7 * blockDim.x];
        idata[tid] = sum;  // Store the sum in shared memory
    }

    __syncthreads();

    // unrolled and optimized reduction using warp shuffle
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            idata[tid] += idata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}