#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // Static shared memory
    __shared__ int smem[DIM];

    // Set thread ID
    unsigned int tid = threadIdx.x;

    // Global index
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // Unrolling 4 blocks with bounds check
    int localSum = 0;
    if (idx < n) localSum += g_idata[idx];
    if (idx + blockDim.x < n) localSum += g_idata[idx + blockDim.x];
    if (idx + 2 * blockDim.x < n) localSum += g_idata[idx + 2 * blockDim.x];
    if (idx + 3 * blockDim.x < n) localSum += g_idata[idx + 3 * blockDim.x];

    // Store local sum in shared memory
    smem[tid] = localSum;
    __syncthreads();

    // In-place reduction in shared memory
    if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];
    __syncthreads();
    if (blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];
    __syncthreads();
    if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];
    __syncthreads();
    if (blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];
    __syncthreads();

    // Unrolling warp reduction
    if (tid < 32)
    {
        // Avoid using volatile for the shared memory
        #pragma unroll
        for(int offset = 32; offset > 0; offset /= 2)
        {
            smem[tid] += smem[tid + offset];
        }
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}