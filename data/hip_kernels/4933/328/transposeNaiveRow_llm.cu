#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeNaiveRow(int *in, int *out, const int nx, const int ny)
{
    // Use shared memory for coalesced access
    __shared__ int tile[TILE_SIZE][TILE_SIZE + 1];

    // Set thread id.
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix < nx && iy < ny)
    {
        // Load data into shared memory
        tile[threadIdx.y][threadIdx.x] = in[iy * nx + ix];
    }
    __syncthreads();

    // Transpose block using shared memory
    unsigned int transposed_ix = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int transposed_iy = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (transposed_ix < ny && transposed_iy < nx)
    {
        // Write transposed data from shared memory
        out[transposed_iy * ny + transposed_ix] = tile[threadIdx.x][threadIdx.y];
    }
}