#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeOffsetUnroll2(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Use warp-level computation with unrolling for better memory efficiency
    if (k + blockDim.x < n)
    {
        float a = A[i];
        float b = B[i];
        C[k] = a + b;

        a = A[i + blockDim.x];
        b = B[i + blockDim.x];
        C[k + blockDim.x] = a + b;
    }
}