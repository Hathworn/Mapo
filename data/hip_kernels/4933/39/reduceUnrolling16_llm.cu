#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling16 (int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 16 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 16;

    // Unrolling 16 with bounds check
    int sum = 0;
    #pragma unroll
    for (int i = 0; i < 16; ++i)
    {
        if (idx + i * blockDim.x < n)
        {
            sum += g_idata[idx + i * blockDim.x];
        }
    }
    
    // Store the sum in the first element of the idata for further reduction
    idata[tid] = sum;
    __syncthreads();

    // In-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        // Synchronize within threadblock
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}