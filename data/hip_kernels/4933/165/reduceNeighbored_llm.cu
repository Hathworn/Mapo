#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighbored(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (idx >= n) return;

    // Convert global data pointer to the local pointer of this block
    extern __shared__ int sdata[];
    sdata[tid] = g_idata[idx];

    // Synchronize within threadblock to ensure all loads are complete
    __syncthreads();

    // In-place reduction in shared memory for better performance
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            sdata[tid] += sdata[tid + stride];
        }
        // Synchronize within threadblock
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}