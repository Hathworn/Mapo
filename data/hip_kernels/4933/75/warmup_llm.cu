#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate global index 
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Ensure within bounds before performing operation
    if (k < n)
    {
        C[k] = A[i] + B[i];
    }
}