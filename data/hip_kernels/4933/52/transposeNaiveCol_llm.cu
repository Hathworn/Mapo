#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNaiveCol(float *out, float *in, const int nx, const int ny)
{
    // Calculate global index
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds to avoid redundant computations
    if (ix < nx && iy < ny)
    {
        // Perform the transpose operation
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}