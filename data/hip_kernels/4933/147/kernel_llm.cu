#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *F, double *D)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to set and test for only tid 0
    __shared__ bool isSet;
    if (threadIdx.x == 0) {
        *F = 12.1f;  // Use literal suffix for float
        *D = 12.1;
        isSet = true; // Set flag in shared memory
    }
    __syncthreads();

    if (isSet && tid == 0) {
        // Task for tid 0 only, after flag is set
    }
}