#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmem(int *g_idata, int *g_odata, unsigned int n)
{
    __shared__ int smem[DIM];

    // Set thread ID
    unsigned int tid = threadIdx.x;

    // Boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Set to smem by each thread
    smem[tid] = idata[tid];
    __syncthreads();

    // In-place reduction in shared memory
    for (int offset = blockDim.x / 2; offset > 32; offset >>= 1) {
        if (tid < offset) smem[tid] += smem[tid + offset];
        __syncthreads();
    }

    // Unrolling the last warp
    if (tid < 32) {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}