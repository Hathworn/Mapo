#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate global thread index
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within matrix bounds
    if (ix < nx && iy < ny) {
        unsigned int idx = iy * nx + ix; // Calculate linear global index
        MatC[idx] = MatA[idx] + MatB[idx]; // Perform element-wise addition
    }
}