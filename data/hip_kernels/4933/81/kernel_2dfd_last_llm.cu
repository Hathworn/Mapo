#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_2dfd_last(float *g_u1, float *g_u2, const int nx, const int iStart, const int iEnd)
{
    // Compute global x index
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute shared memory index and global memory index for starting point
    unsigned int stx = threadIdx.x + NPAD;
    unsigned int idx = ix + iStart * nx;

    // Shared memory buffer with padding for stencil computation
    __shared__ float tile[BDIMX + NPAD2];

    const float alpha = 0.12f;

    // Registers to hold y-dimension values for stencil computation
    float yval[9];

    // Load initial values into yval array
    #pragma unroll
    for (int i = 0; i < 8; i++) yval[i] = g_u2[idx + (i - 4) * nx];

    // Precompute index offset for y-dimension loop
    int iskip = NPAD * nx;

    // Loop over y-dimension in the global grid
    #pragma unroll 9
    for (int iy = iStart; iy < iEnd; iy++)
    {
        // Load next y value
        yval[8] = g_u2[idx + iskip];

        // Load boundary values into shared memory using thread synchronization
        if(threadIdx.x < NPAD)
        {
            tile[threadIdx.x] = g_u2[idx - NPAD];
            tile[stx + BDIMX] = g_u2[idx + BDIMX];
        }
        
        // Main data loaded into shared memory
        tile[stx] = yval[4];
        
        // Synchronize to ensure all data is loaded
        __syncthreads();

        // Ensure within proper bounds before computation
        if (ix >= NPAD && ix < nx - NPAD)
        {
            // Implement 8-th order finite difference stencil
            float tmp = coef[0] * tile[stx] * 2.0f;

            #pragma unroll
            for(int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (tile[stx - d] + tile[stx + d]);
            }

            #pragma unroll
            for(int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (yval[4 - d] + yval[4 + d]);
            }

            // Update time dimension
            g_u1[idx] = yval[4] + yval[4] - g_u1[idx] + alpha * tmp;
        }

        // Shift y-values up for next iteration
        #pragma unroll 8
        for (int i = 0; i < 8; i++)
        {
            yval[i] = yval[i + 1];
        }

        // Advance to next row in global index
        idx += nx;
        
        // Synchronization to prevent race conditions
        __syncthreads();
    }
}