#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysZeroCopyWithUVAOffset(float *A, float *B, float *C, const int N, int offset)
{
    // Cache the global thread index to reduce repeated calculations
    int i = blockIdx.x * blockDim.x + threadIdx.x + offset;
    
    // Precompute limit for faster comparison
    int limit = N + offset;

    // Check condition using precomputed limit
    if (i < limit) {
        C[i] = A[i] + B[i];
    }
}