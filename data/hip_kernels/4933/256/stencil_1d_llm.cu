#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil_1d(float *in, float *out)
{
    // shared memory
    __shared__ float smem[BDIM + 2 * RADIUS];

    // index to global memory
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // index to shared memory for stencil calculation
    int sidx = threadIdx.x + RADIUS;

    // Load center element from global to shared memory
    smem[sidx] = in[idx];

    // Load halo elements to shared memory
    if (threadIdx.x < RADIUS)
    {
        smem[threadIdx.x] = in[idx - RADIUS]; // Load left halo
        smem[threadIdx.x + BDIM + RADIUS] = in[idx + BDIM]; // Load right halo
    }

    // Synchronize (ensure all the data is available)
    __syncthreads();

    // Apply the stencil
    float tmp = 0.0f;
    #pragma unroll
    for (int i = 1; i <= RADIUS; i++)
    {
        tmp += coef[i] * (smem[sidx + i] - smem[sidx - i]);
    }

    // Store the result
    out[idx] = tmp;
}