#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighboredLess(int *g_idata, int *g_odata, unsigned int n)
{
    // Set the thread id and block id
    unsigned int tid = threadIdx.x;
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Convert global data pointer to the local pointer of this block.
    int *idata = g_idata + blockIdx.x * blockDim.x;
    
    // Boundary check.
    if (idx >= n) return;
    
    // Loop unrolling for efficiency
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        // Use volatile to prevent compiler optimization issues
        volatile int* v_idata = idata;
        
        // Convert tid into local array index.
        int index = 2 * stride * tid;
        
        if (index < blockDim.x)
        {
            v_idata[index] += v_idata[index + stride];
        }

        // Synchronize within threadblock.
        __syncthreads();
    }

    // Write result for this block to global memory.
    if (tid == 0)
    {
        g_odata[blockIdx.x] = idata[0];
    }
}