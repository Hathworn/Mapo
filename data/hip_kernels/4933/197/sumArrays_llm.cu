#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop unrolling to process multiple elements per thread
    int stride = blockDim.x * gridDim.x;
    for (; i < N; i += stride)
    {
        C[i] = A[i] + B[i];
        if (i + 1 < N)
            C[i + 1] = A[i + 1] + B[i + 1];
        // Add more unrolling if the GPU can handle more work per thread
    }
}