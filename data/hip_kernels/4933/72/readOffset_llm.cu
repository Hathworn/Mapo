#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate the unique global index for each thread
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Check if calculated index is within bounds before performing calculation
    if (k < n) {
        C[i] = A[k] + B[k];
    }
}