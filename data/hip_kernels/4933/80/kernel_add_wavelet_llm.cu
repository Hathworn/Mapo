#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_add_wavelet (float *g_u2, float wavelets, const int nx, const int ny, const int ngpus) {
    // Calculate global x index and only proceed if within bounds
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix < nx) {
        // Replaced if condition with a ternary operator for clarity and efficiency
        int ipos = (ngpus == 2) ? (ny - 10) : (ny / 2 - 10);
        unsigned int idx = ipos * nx + ix;
        
        // Optimize to avoid unnecessary memory access
        if(ix == nx / 2) {
            atomicAdd(&g_u2[idx], wavelets); // Use atomicAdd to prevent race conditions
        }
    }
}