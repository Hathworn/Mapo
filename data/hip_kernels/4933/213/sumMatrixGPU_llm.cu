#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate global thread ID
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate the linear index
    unsigned int idx = iy * nx + ix;

    // Perform addition within valid range
    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}