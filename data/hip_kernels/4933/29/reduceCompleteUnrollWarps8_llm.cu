#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceCompleteUnrollWarps8 (int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // Unrolling 8 with boundary check optimization
    if (idx + 7 * blockDim.x < n)
    {
        int sum = 0;
        #pragma unroll
        for (int i = 0; i < 8; ++i)
        {
            sum += g_idata[idx + i * blockDim.x];
        }
        g_idata[idx] = sum;
    }

    __syncthreads();

    // In-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();

    // Unrolling warp using volatile to ensure memory consistency within the warp
    if (tid < 32)
    {
        volatile int *vsmem = idata;

        // Manual unrolling of loop to minimize divergence
        if (blockDim.x >= 64) vsmem[tid] += vsmem[tid + 32];
        if (blockDim.x >= 32) vsmem[tid] += vsmem[tid + 16];
        if (blockDim.x >= 16) vsmem[tid] += vsmem[tid + 8];
        if (blockDim.x >= 8)  vsmem[tid] += vsmem[tid + 4];
        if (blockDim.x >= 4)  vsmem[tid] += vsmem[tid + 2];
        if (blockDim.x >= 2)  vsmem[tid] += vsmem[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}