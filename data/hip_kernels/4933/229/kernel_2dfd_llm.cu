#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_2dfd(float *g_u1, float *g_u2, const int nx, const int iStart, const int iEnd)
{
    // Global to line index
    unsigned int ix  = blockIdx.x * blockDim.x + threadIdx.x;

    // Smem idx for current point
    unsigned int stx = threadIdx.x + NPAD;
    unsigned int idx  = ix + iStart * nx;

    // Shared memory for x dimension
    __shared__ float line[BDIMX + NPAD2];

    // A coefficient related to physical properties
    const float alpha = 0.12f;

    // Register for y value
    float yval[9];

    // Initialize yval
    #pragma unroll 8
    for (int i = 0; i < 8; i++) yval[i] = g_u2[idx + (i - 4) * nx];

    // Skip for the bottom most y value
    int iskip = NPAD * nx;

    // Optimize loop for better ILP (Instruction Level Parallelism)
    for (int iy = iStart; iy < iEnd; iy++)
    {
        // Get yval[8] here
        yval[8] = g_u2[idx + iskip];

        // Read halo part using one branch
        if(threadIdx.x < NPAD)
        {
            line[threadIdx.x]  = g_u2[idx - NPAD];
            line[stx + BDIMX]    = g_u2[idx + BDIMX];
        }
        line[stx] = yval[4];
        __syncthreads();

        // 8rd fd operator, utilize more registers for intermediate calculations
        if ( (ix >= NPAD) && (ix < nx - NPAD) )
        {
            // Center point
            float tmp = coef[0] * line[stx] * 2.0f;

            #pragma unroll
            for(int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (line[stx - d] + line[stx + d]);
            }

            #pragma unroll
            for(int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (yval[4 - d] + yval[4 + d]);
            }

            // Time dimension update, separated for better latency hiding
            g_u1[idx] = yval[4] + yval[4] - g_u1[idx] + alpha * tmp;
        }

        // Update yval using unrolling
        #pragma unroll 8
        for (int i = 0; i < 8 ; i++)
        {
            yval[i] = yval[i + 1];
        }

        // Advance on global idx
        idx  += nx;
        __syncthreads();
    }
}