#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmem(int *g_idata, int *g_odata, unsigned int n)
{
    __shared__ int smem[DIM];

    // Set thread ID
    unsigned int tid = threadIdx.x;

    // Boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Set to smem by each thread
    smem[tid] = (idx < n) ? idata[tid] : 0; // Check boundary condition
    __syncthreads();

    // In-place reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) smem[tid] += smem[tid + s]; // Reduce within block
        __syncthreads();
    }

    // Unrolling warp
    if (tid < 32) {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}