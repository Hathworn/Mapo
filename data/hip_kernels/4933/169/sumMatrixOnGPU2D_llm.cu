#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize using a shared memory buffer for better memory access coalescing
__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, int NX, int NY)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Early exit if the index is out of range
    if (ix >= NX || iy >= NY) return;

    unsigned int idx = iy * NX + ix;

    C[idx] = A[idx] + B[idx];
}