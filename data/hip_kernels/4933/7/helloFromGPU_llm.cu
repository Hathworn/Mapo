#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU()
{
    // Use one-dimensional grid and block index calculation
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Print message with unique thread identifier
    printf("Hello from GPU! UniqueThreadID: %d.\n", idx);
}