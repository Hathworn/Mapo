#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll4Col(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = (blockDim.x * blockIdx.x + threadIdx.x) * 4; // Improved access pattern
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        unsigned int ti = iy * nx + ix; // Access in rows
        unsigned int to = ix * ny + iy; // Access in columns

        // Unrolled memory transfer
        out[ti]                = in[to];
        out[ti +   blockDim.x] = in[to +   blockDim.x * ny];
        out[ti + 2 * blockDim.x] = in[to + 2 * blockDim.x * ny];
        out[ti + 3 * blockDim.x] = in[to + 3 * blockDim.x * ny];
    }
}