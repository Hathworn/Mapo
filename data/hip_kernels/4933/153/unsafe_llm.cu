#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unsafe(int *shared_var, int *values_read, int N, int iters)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= N) return;

    __shared__ int shared_buf;
    if (threadIdx.x == 0) {
        shared_buf = *shared_var; // Use shared memory for faster access during the loop
    }
    __syncthreads();

    values_read[tid] = atomicAdd(&shared_buf, 1); // Atomic operation to prevent race conditions

    for (int i = 0; i < iters; i++)
    {
        atomicAdd(&shared_buf, 1); // Atomic operation in the loop
    }

    if (threadIdx.x == 0) {
        atomicAdd(shared_var, shared_buf); // Write back to global memory from shared memory
    }
}