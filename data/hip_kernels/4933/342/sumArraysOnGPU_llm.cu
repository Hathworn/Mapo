#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, int n)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread is within array bounds
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}