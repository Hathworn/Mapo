#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling4(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 4;

    // Unrolling 4
    int sum = 0;
    if (idx + 3 * blockDim.x < n)
    {
        sum = g_idata[idx] + g_idata[idx + blockDim.x] +
              g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x];
    }
    // Store computed sum back to idata
    idata[tid] = sum;

    __syncthreads();

    // In-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }

        // Synchronize within threadblock
        __syncthreads();
    }

    // Write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}