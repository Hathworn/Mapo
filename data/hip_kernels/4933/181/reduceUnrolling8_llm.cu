#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling8(int *g_idata, int *g_odata, unsigned int n) {
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // Check bounds and perform unrolled reduction
    int sum = 0;
    if (idx < n) {
        sum += g_idata[idx];
        if (idx + blockDim.x < n) sum += g_idata[idx + blockDim.x];
        if (idx + 2 * blockDim.x < n) sum += g_idata[idx + 2 * blockDim.x];
        if (idx + 3 * blockDim.x < n) sum += g_idata[idx + 3 * blockDim.x];
        if (idx + 4 * blockDim.x < n) sum += g_idata[idx + 4 * blockDim.x];
        if (idx + 5 * blockDim.x < n) sum += g_idata[idx + 5 * blockDim.x];
        if (idx + 6 * blockDim.x < n) sum += g_idata[idx + 6 * blockDim.x];
        if (idx + 7 * blockDim.x < n) sum += g_idata[idx + 7 * blockDim.x];
    }
    idata[tid] = sum;

    __syncthreads();

    // In-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads(); // Synchronize within threadblock
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}