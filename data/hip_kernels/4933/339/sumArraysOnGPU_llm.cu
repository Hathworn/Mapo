#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, int n)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we do not exceed the array bounds
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}