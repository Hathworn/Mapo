#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Use 2D thread indexing for better readability and flexibility
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix < nx && iy < ny) {
        unsigned int idx = iy * nx + ix;
        // Sum the corresponding elements
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}