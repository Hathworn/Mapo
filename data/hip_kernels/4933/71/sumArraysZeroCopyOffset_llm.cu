#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysZeroCopyOffset(float *A, float *B, float *C, const int N, int offset)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Move condition check inside to minimize branch divergence
    if (i + offset < N) 
    {
        C[i + offset] = A[i + offset] + B[i + offset];
    }
}