#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void refill_randoms(float *dRand, int N, hiprandState *states)
{
int i;
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int nthreads = gridDim.x * blockDim.x;
hiprandState *state = states + tid;

for (i = tid; i < N; i += nthreads)
{
dRand[i] = hiprand_uniform(state);
}
}