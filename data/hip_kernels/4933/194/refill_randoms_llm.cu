#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refill_randoms(float *dRand, int N, hiprandState *states)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;
    hiprandState localState = states[tid];  // Load state to a local variable for efficiency

    for (int i = tid; i < N; i += nthreads)
    {
        dRand[i] = hiprand_uniform(&localState);
    }

    states[tid] = localState;  // Store state back to global memory
}