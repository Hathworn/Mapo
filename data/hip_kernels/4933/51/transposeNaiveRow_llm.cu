#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNaiveRow(float *out, float *in, const int nx, const int ny)
{
    // Utilize shared memory to reduce global memory access
    __shared__ float tile[32][32 + 1]; // Padding to prevent bank conflicts

    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        tile[threadIdx.y][threadIdx.x] = in[iy * nx + ix];
    }
    __syncthreads();

    ix = blockIdx.y * blockDim.y + threadIdx.x; // Transpose access pattern
    iy = blockIdx.x * blockDim.x + threadIdx.y;

    if (ix < ny && iy < nx)
    {
        out[iy * ny + ix] = tile[threadIdx.x][threadIdx.y];
    }
}