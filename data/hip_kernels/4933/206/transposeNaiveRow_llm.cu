#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNaiveRow(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Use shared memory to reduce global memory accesses
    __shared__ float tile[32][33]; // Avoid bank conflicts

    if (ix < nx && iy < ny)
    {
        // Read the matrix tile into shared memory
        tile[threadIdx.y][threadIdx.x] = in[iy * nx + ix];
        __syncthreads();

        // Write the transposed matrix tile to global memory
        out[ix * ny + iy] = tile[threadIdx.y][threadIdx.x];
    }
}
```
