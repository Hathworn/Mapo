#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void printThreadIndex(int *A, const int nx, const int ny)
{
    // Calculate global thread index
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Early exit if thread is out of bounds
    if (ix >= nx || iy >= ny) return;

    // Compute global linear index
    unsigned int idx = iy * nx + ix;

    // Print information
    printf("thread_id (%d,%d) block_id (%d,%d) coordinate (%d,%d) global index"
           " %2d ival %2d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y,
           ix, iy, idx, A[idx]);
}