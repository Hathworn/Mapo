#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighboredSmem(int *g_idata, int *g_odata, unsigned int n)
{
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    // load data to shared memory
    smem[tid] = idata[tid];
    __syncthreads();

    // in-place reduction in shared memory; stride doubling
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            smem[tid] += smem[tid + stride];
        }
        __syncthreads(); // synchronize within threadblock
    }

    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}