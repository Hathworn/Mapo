#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_add_wavelet (float *g_u2, float wavelets, const int nx, const int ny, const int ngpus)
{
    // Calculate position offset only once outside loop
    int ipos = (ngpus == 2 ? ny - 10 : ny / 2 - 10); 

    // Compute global index
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate the overall index
    unsigned int idx = ipos * nx + ix;

    // Use atomic add for safer addition on the same index
    if (ix == nx / 2) {
        atomicAdd(&g_u2[idx], wavelets);
    }
}