#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceInterleaved(int *g_idata, int *g_odata, unsigned int n)
{
    // Calculate global and local thread index
    unsigned int tid = threadIdx.x;
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if (idx >= n) return;

    // Perform initial contribution to local memory to ensure coalesced access
    if (idx < n) idata[tid] = g_idata[idx]; 
    __syncthreads();

    // In-place reduction in shared (local) memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global mem
    if (tid == 0)
    {
        g_odata[blockIdx.x] = idata[0];
    }
}