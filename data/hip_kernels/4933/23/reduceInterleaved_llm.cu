#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceInterleaved(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if (idx >= n) return;

    // In-place reduction in global memory
    for (int stride = 1; stride < blockDim.x; stride *= 2) // Optimized loop start from 1
    {
        // Check that stride element is within bounds 
        if (tid % (2 * stride) == 0 && tid + stride < blockDim.x) // Ensure no out of bounds access
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads(); // Synchronize threads in each step
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}