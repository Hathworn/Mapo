#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll4Col(float *out, float *in, const int nx, const int ny)
{
    // Calculate thread's global indices
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Compute indexes for input and output with row and column access
    unsigned int ti = iy * nx + ix; 
    unsigned int to = ix * ny + iy; 

    // Check boundaries and perform loop unrolling for optimized memory access
    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        #pragma unroll 4
        for (int i = 0; i < 4; i++)
        {
            unsigned int idx_out = ti + i * blockDim.x;
            unsigned int idx_in = to + i * blockDim.x * ny;
            out[idx_out] = in[idx_in];
        }
    }
}