#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    
    // Ensure index is within bounds for both dimensions before computation
    if (ix < nx && iy < ny) 
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}