#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeDiagonalCol(float *out, float *in, const int nx, const int ny)
{
    // Calculate the linear index for blocks and threads
    unsigned int blk_id = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int th_id = threadIdx.y * blockDim.x + threadIdx.x;

    unsigned int ix = blk_id % gridDim.x * blockDim.x + threadIdx.x;
    unsigned int iy = blk_id / gridDim.x * blockDim.y + threadIdx.y;

    // Perform transpose only within bounds
    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}