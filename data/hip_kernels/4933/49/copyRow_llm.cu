#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyRow(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockIdx.y;  // Removed unnecessary blockDim.y calculation since only rows are copied

    if (ix < nx && iy < ny)
    {
        unsigned int idx = iy * nx + ix;  // Calculate the index once
        out[idx] = in[idx];
    }
}