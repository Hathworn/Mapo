#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighbored(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if (idx >= n) return;

    // In-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) // Optimize loop: reduce iterations
    {
        __syncthreads(); // Ensure complete previous step before continuing

        if (tid < stride) // Only threads within stride perform operations
        {
            idata[tid] += idata[tid + stride];
        }
    }

    // Ensure all operations complete before writing result
    __syncthreads(); 

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}