#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Use 2D indexing to clarify matrix element access
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < nx && iy < ny) // Ensure threads are within matrix bounds
    {
        unsigned int idx = iy * nx + ix; // Compute flattened index
        MatC[idx] = MatA[idx] + MatB[idx]; // Perform element-wise addition
    }
}