#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void standard_kernel(float a, float *out, int iters)
{
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

    if (tid == 0)
    {
        // Compute power outside the loop as it is independent of loop iterations
        float tmp = powf(a, 2.0f);

        // Set result to output pointer
        *out = tmp;
    }
}