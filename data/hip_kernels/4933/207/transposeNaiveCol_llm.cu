#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNaiveCol(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Utilize shared memory for data movement
    __shared__ float tile[32][32]; // Assuming block size is 32x32

    if (ix < nx && iy < ny)
    {
        tile[threadIdx.y][threadIdx.x] = in[ix * ny + iy];
    }

    __syncthreads(); // Ensure all tiles are loaded

    // Calculate transposed indices
    unsigned int transposed_ix = blockDim.y * blockIdx.y + threadIdx.x;
    unsigned int transposed_iy = blockDim.x * blockIdx.x + threadIdx.y;

    if (transposed_ix < ny && transposed_iy < nx)
    {
        out[transposed_iy * ny + transposed_ix] = tile[threadIdx.x][threadIdx.y];
    }
}