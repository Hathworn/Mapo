#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_reduction(int* shared_var, int* input_values, int N, int iters) {
    __shared__ int local_mem[256];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    for (int iter = 0; iter < iters; iter++) {
        if (tid < N) {
            local_mem[local_tid] = input_values[tid];
        } else {
            local_mem[local_tid] = 0; // Ensure valid data for threads beyond N
        }

        __syncthreads(); // Synchronize to ensure all loads are complete

        // Use parallel reduction within shared memory
        for (int stride = 1; stride < blockDim.x; stride *= 2) {
            int index = 2 * stride * local_tid;
            if (index < blockDim.x) {
                local_mem[index] += local_mem[index + stride];
            }
            __syncthreads(); // Synchronize to ensure stride computation is complete
        }

        if (local_tid == 0) {
            atomicAdd(shared_var, local_mem[0]); // Only sum from thread 0 is added to shared var
        }
    }
}