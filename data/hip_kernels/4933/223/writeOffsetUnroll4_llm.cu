#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeOffsetUnroll4(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * (blockDim.x * 4) + threadIdx.x; // Load multiple elements per thread
    unsigned int k = i + offset;

    if (k + 3 * blockDim.x < n) // Efficient boundary check
    {
        float4 a, b, c; // Use float4 for coalesced memory access
        a = *((float4*)&A[i]);
        b = *((float4*)&B[i]);
        c.x = a.x + b.x;
        c.y = a.y + b.y;
        c.z = a.z + b.z;
        c.w = a.w + b.w;
        *((float4*)&C[k]) = c;
    }
}