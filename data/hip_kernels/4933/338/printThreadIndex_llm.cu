#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void printThreadIndex(int *A, const int nx, const int ny)
{
    // Calculate unique thread index using block and thread indices
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure the thread is within matrix bounds
    if (ix < nx && iy < ny)
    {
        unsigned int idx = iy * nx + ix;
        
        // Print thread information and array value
        printf("thread_id: (%d, %d), block_id: (%d, %d), coordinate: (%d, %d) global index %2d ival %2d\n",
               threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx, A[idx]);
    }
}