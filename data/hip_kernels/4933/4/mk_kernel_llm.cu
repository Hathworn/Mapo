#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mk_kernel(char* keep_mem, size_t bytes)
{
    // Calculate global thread index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use each thread to initialize one element if within bounds
    if (idx < bytes)
    {
        keep_mem[idx] = 0;
    }
}