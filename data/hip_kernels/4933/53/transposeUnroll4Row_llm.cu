#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll4Row(float *out, float *in, const int nx, const int ny)
{
    // Calculate initial x and y indices
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Calculate linear memory indices
    unsigned int ti = iy * nx + ix; 
    unsigned int to = ix * ny + iy; 

    // Ensure within bounds
    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        // Perform optimized memory transactions
        #pragma unroll // Unroll loop for efficiency
        for (int i = 0; i < 4; ++i) {
            out[to + i * ny * blockDim.x] = in[ti + i * blockDim.x];
        }
    }
}