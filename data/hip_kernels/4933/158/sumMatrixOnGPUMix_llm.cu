#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate thread index using efficient 2D grid layout
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    // Boundary check before accessing memory
    if (ix < nx && iy < ny) {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}