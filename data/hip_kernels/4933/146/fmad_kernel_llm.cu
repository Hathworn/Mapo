#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fmad_kernel(double x, double y, double *out)
{
    // Use only one thread to compute the operation
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        *out = __dmul_rd(x, x) + y; // Use multiply-add intrinsic for better performance
    }
}