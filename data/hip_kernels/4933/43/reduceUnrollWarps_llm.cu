#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrollWarps(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // unrolling 2
    if (idx + blockDim.x < n) g_idata[idx] += g_idata[idx + blockDim.x];

    // synchronize before in-place reduction
    __syncthreads();

    // optimized reduction using loop unrolling
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // unrolling last warp with volatile
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}