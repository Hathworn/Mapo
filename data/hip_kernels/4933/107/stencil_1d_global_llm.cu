#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil_1d_global(float* in, float* out, float* dcoef) {
    // Shared memory
    __shared__ float smem[BDIM + 2 * RADIUS];

    // Index to global memory
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Index to shared memory for stencil calculation
    int sidx = threadIdx.x + RADIUS;

    // Read data from global memory into shared memory
    smem[sidx] = in[idx];

    // Read halo part to shared memory
    if (threadIdx.x < RADIUS) {
        smem[sidx - RADIUS] = in[idx - RADIUS];
        smem[sidx + BDIM] = in[idx + BDIM];
    }

    // Synchronize (ensure all the data is available)
    __syncthreads();

    // Apply the stencil
    float tmp = 0.0f;
    
    // Unroll the loop for better performance
    #pragma unroll RADIUS  // Unroll pragma applies directly to the loop
    for (int i = 1; i <= RADIUS; i++) {
        tmp += dcoef[i] * (smem[sidx + i] - smem[sidx - i]);
    }

    // Store the result
    out[idx] = tmp;
}