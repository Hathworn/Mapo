#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readWriteOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate global index with included offset
    unsigned int k = blockIdx.x * blockDim.x + threadIdx.x + offset;

    // Ensure the index is within the bounds
    if (k < n) {
        // Perform addition operation
        C[k] = A[k] + B[k];
    }
}