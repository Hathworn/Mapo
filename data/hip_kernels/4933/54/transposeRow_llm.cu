#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeRow(float *out, float *in, const int nx, const int ny)
{
    // Calculate global row and column indices.
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure indices are within the bounds of the input matrix.
    if (ix < nx && iy < ny)
    {
        int input_index = iy * nx + ix; // Calculate 1D index for the input array.
        int output_index = ix * ny + iy; // Calculate 1D index for the transposed output array.
        out[output_index] = in[input_index]; // Perform the transpose by swapping indices.
    }
}