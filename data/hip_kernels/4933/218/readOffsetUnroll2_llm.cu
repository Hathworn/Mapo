#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readOffsetUnroll2(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate the initial index with offset
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    unsigned int k = i + offset;

    // Use shared memory for potential speedup if applicable
    if (k < n) {
        C[i] = A[k] + B[k];
    }
    if (k + blockDim.x < n) {
        C[i + blockDim.x] = A[k + blockDim.x] + B[k + blockDim.x];
    }
}