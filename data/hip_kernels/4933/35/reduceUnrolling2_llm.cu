#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling2(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + tid;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // unrolling 2 with boundary check
    if (idx + blockDim.x < n) {
        g_idata[idx] += g_idata[idx + blockDim.x];
    }
    __syncthreads();

    // optimize in-place reduction using warp shuffle for small strides
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}