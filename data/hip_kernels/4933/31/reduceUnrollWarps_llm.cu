#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrollWarps (int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // Unrolling 2
    if (idx + blockDim.x < n) {
        g_idata[idx] += g_idata[idx + blockDim.x];
    }
    __syncthreads();

    // In-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads(); // Synchronize within threadblock
    }

    // Unrolling last warp
    if (tid < 32) {
        volatile int *vsmem = idata;
        #pragma unroll // Unroll warp level computation for optimization
        for (int offset = 32; offset > 0; offset >>= 1) {
            vsmem[tid] += vsmem[tid + offset];
        }
    }

    // Output the result for this block to g_odata
    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}