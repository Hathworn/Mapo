#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling8(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // Unrolling 8
    if (idx + 7 * blockDim.x < n)
    {
        // Load inputs into registers and perform unrolled reduction
        int sum = g_idata[idx] + g_idata[idx + blockDim.x] +
                  g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x] +
                  g_idata[idx + 4 * blockDim.x] + g_idata[idx + 5 * blockDim.x] +
                  g_idata[idx + 6 * blockDim.x] + g_idata[idx + 7 * blockDim.x];
        idata[tid] = sum;
    }
    else if (idx < n) // Handle remaining elements
    {
        idata[tid] = g_idata[idx];
    }
    else
    {
        idata[tid] = 0; // Handle out-of-bounds
    }

    __syncthreads();

    // In-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }

        // Synchronize within threadblock
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}