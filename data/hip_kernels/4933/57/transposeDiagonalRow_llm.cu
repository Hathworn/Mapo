#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeDiagonalRow(float *out, float *in, const int nx, const int ny)
{
    // Calculate unique block index for diagonal traversal
    unsigned int blk_x = blockIdx.x;
    unsigned int blk_y = blockIdx.y;

    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    // Ensure within bounds before writing
    if (ix < nx && iy < ny)
    {
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}