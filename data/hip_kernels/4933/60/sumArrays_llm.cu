#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop to improve performance
    if (i < N) {
        C[i] = A[i] + B[i];
        if (i + blockDim.x < N) C[i + blockDim.x] = A[i + blockDim.x] + B[i + blockDim.x];
        if (i + 2 * blockDim.x < N) C[i + 2 * blockDim.x] = A[i + 2 * blockDim.x] + B[i + 2 * blockDim.x];
        if (i + 3 * blockDim.x < N) C[i + 3 * blockDim.x] = A[i + 3 * blockDim.x] + B[i + 3 * blockDim.x];
    }
}