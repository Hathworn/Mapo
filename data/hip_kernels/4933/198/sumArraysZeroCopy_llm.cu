#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysZeroCopy(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check against boundary and process multiple elements per thread to optimize workload
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        C[idx] = A[idx] + B[idx];
    }
}