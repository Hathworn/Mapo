#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate the unique thread ID
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure the thread is inside the bounds
    if (ix < nx && iy < ny)
    {
        // Calculate the linear index within the matrix
        unsigned int idx = iy * nx + ix;
        
        // Compute and store the sum of sines
        MatC[idx] = sin(MatA[idx]) + sin(MatB[idx]);
    }
}