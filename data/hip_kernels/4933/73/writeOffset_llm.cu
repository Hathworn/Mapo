#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate global thread index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Check bounds and compute only if within limits
    if (k < n) {
        C[k] = A[i] + B[i];
    }
}