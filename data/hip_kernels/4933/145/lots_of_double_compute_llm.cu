#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lots_of_double_compute(double *inputs, int N, size_t niters, double *outputs)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t nthreads = gridDim.x * blockDim.x;

    // Iterate over threads with stride
    for ( ; tid < N; tid += nthreads)
    {
        double val = inputs[tid]; // Cache input value in register

        // Unroll loop for better performance
        for (size_t iter = 0; iter < niters; iter++)
        {
            val = ((val + 5.0 - 101.0) / 3.0 + 102.0 + 1.07 - 103.0) / 1.037 + 104.0;
            val = ((val + 3.00 - 105.0) / 0.22 + 106.0);
        }

        outputs[tid] = val; // Store result
    }
}