#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refill_randoms(float *dRand, int N, hiprandState *states)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Precompute stride

    hiprandState localState = states[tid]; // Load state into local variable

    for (int i = tid; i < N; i += stride)
    {
        dRand[i] = hiprand_uniform(&localState); // Use local copy of state
    }

    states[tid] = localState; // Save modified state back
}