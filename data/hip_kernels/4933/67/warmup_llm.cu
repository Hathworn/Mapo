#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function
__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Only perform operations if k is less than n
    if (k < n) {
        C[k] = A[i] + B[i]; // Perform vector addition
    }
}