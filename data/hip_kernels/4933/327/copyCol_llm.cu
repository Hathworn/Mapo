#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyCol(int *in, int *out, const int nx, const int ny)
{
    // Combine ix and iy calculation for cleaner code
    unsigned int id = threadIdx.x + blockIdx.x * blockDim.x + 
                      (threadIdx.y + blockIdx.y * blockDim.y) * nx;

    // Use one calculation to check bounds
    if (id < nx * ny)
    {
        out[id] = in[id];
    }
}