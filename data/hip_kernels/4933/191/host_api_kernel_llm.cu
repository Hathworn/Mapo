#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void host_api_kernel(float *randomValues, float *out, int N)
{
    // Get thread index with minor optimization using block-wide stride
    int tid = (blockIdx.x * blockDim.x + threadIdx.x);
    int stride = blockDim.x * gridDim.x; // Calculate total stride

    for (int i = tid; i < N; i += stride)
    {
        float rand = randomValues[i];
        rand = rand * 2;
        out[i] = rand;
    }
}