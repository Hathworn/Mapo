#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceSmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // Static shared memory
    __shared__ int smem[DIM];

    // Set thread ID
    unsigned int tid = threadIdx.x;

    // Global index, 4 blocks of input data processed at a time
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // Unrolling 4 blocks
    int tmpSum = 0;

    // Boundary check and sum up unrolled data
    if (idx < n) tmpSum = g_idata[idx] 
                       + ((idx + blockDim.x < n) ? g_idata[idx + blockDim.x] : 0)
                       + ((idx + 2 * blockDim.x < n) ? g_idata[idx + 2 * blockDim.x] : 0)
                       + ((idx + 3 * blockDim.x < n) ? g_idata[idx + 3 * blockDim.x] : 0);

    smem[tid] = tmpSum;
    __syncthreads();

    // In-place reduction in shared memory with loop unrolling
    #pragma unroll
    for (int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) smem[tid] += smem[tid + s];
        __syncthreads();
    }

    // Unrolling last warp without __syncthreads
    if (tid < 32) {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // Write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}