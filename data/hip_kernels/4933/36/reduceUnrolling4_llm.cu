#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling4 (int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 4;

    // Unrolling 4
    int sum = 0;
    if (idx + 3 * blockDim.x < n) {
        sum = g_idata[idx] + g_idata[idx + blockDim.x] + g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x];
    } else {
        // Handle remaining elements if any
        if (idx < n) sum += g_idata[idx];
        if (idx + blockDim.x < n) sum += g_idata[idx + blockDim.x];
        if (idx + 2 * blockDim.x < n) sum += g_idata[idx + 2 * blockDim.x];
    }
    idata[tid] = sum;

    __syncthreads();

    // In-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        // Synchronize within threadblock
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}