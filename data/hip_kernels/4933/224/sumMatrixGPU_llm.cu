#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate the global index for the thread
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure the thread operates within matrix bounds
    if (ix < nx && iy < ny)
    {
        unsigned int idx = iy * nx + ix; // Compute the flat index
        MatC[idx] = MatA[idx] + MatB[idx]; // Perform element-wise addition
    }
}