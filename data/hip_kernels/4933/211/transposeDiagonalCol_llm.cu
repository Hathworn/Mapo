#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeDiagonalCol(float *out, float *in, const int nx, const int ny)
{
    // Calculate global thread (work item) indices
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Utilize transposed grid launch for better memory coalescing
    unsigned int threadIdx = iy * nx + ix;

    // Check boundaries to prevent out-of-bounds access
    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}