#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iKernel(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure boundary check and leverage memory coalescing
    if (i < N) {
        float a_val = A[i];  // Load into register
        float b_val = B[i];  // Load into register
        C[i] = a_val + b_val;  // Use registers for computation
    }
}