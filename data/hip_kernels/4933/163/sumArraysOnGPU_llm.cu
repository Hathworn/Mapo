#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Compute stride for cooperative threads

    for (; i < N; i += stride) // Use loop to handle larger arrays
    {
        C[i] = A[i] + B[i];
    }
}