#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Only compute if index is within bounds
    if (idx < N)
    {
        // Perform the addition once per index
        C[idx] = A[idx] + B[idx];
    }
}