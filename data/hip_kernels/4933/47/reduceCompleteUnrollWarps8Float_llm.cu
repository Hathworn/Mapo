#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceCompleteUnrollWarps8Float (float *g_idata, float *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    
    // Perform unrolling for accumulations
    float sum = 0;
    if (idx + 7 * blockDim.x < n)
    {
        sum += g_idata[idx];
        sum += g_idata[idx + blockDim.x];
        sum += g_idata[idx + 2 * blockDim.x];
        sum += g_idata[idx + 3 * blockDim.x];
        sum += g_idata[idx + 4 * blockDim.x];
        sum += g_idata[idx + 5 * blockDim.x];
        sum += g_idata[idx + 6 * blockDim.x];
        sum += g_idata[idx + 7 * blockDim.x];
    }

    // Store sum in shared memory
    __shared__ float sdata[1024];
    sdata[tid] = sum;
    __syncthreads();
    
    // In-place reduction using complete unrolling
    if (blockDim.x >= 1024 && tid < 512) sdata[tid] += sdata[tid + 512];
    __syncthreads();
    if (blockDim.x >= 512 && tid < 256) sdata[tid] += sdata[tid + 256];
    __syncthreads();
    if (blockDim.x >= 256 && tid < 128) sdata[tid] += sdata[tid + 128];
    __syncthreads();
    if (blockDim.x >= 128 && tid < 64) sdata[tid] += sdata[tid + 64];
    __syncthreads();

    // Unrolling warp
    if (tid < 32)
    {
        volatile float *vsmem = sdata; // Use volatile to avoid compiler optimizations
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}