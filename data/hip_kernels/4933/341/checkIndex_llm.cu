#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void checkIndex(void)
{
    // Using a single printf call for improved efficiency
    printf("tIdx: (%d, %d, %d), bIdx: (%d, %d, %d), bDim: (%d, %d, %d), gDim: (%d, %d, %d)\n",
           threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z,
           blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}