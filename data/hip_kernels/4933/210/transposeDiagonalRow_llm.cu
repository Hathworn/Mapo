#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeDiagonalRow(float *out, float *in, const int nx, const int ny)
{
    unsigned int blk_y = blockIdx.y; // Use blockIdx.y directly for better grid utilization
    unsigned int blk_x = blockIdx.x; // Use blockIdx.x directly for better grid utilization

    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}