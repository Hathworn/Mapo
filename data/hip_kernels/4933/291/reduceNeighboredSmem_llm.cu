#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighboredSmem(int *g_idata, int *g_odata, unsigned int n)
{
    __shared__ int smem[DIM];
    
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + tid;
    
    // Check if index is within bounds and load data into shared memory
    if (idx < n) {
        smem[tid] = g_idata[idx];
    } else {
        smem[tid] = 0;
    }
    __syncthreads();
    
    // Perform in-place reduction using shared memory
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        // Ensure stride doesn't access out of bound memory
        if (tid % (2 * stride) == 0 && (tid + stride) < blockDim.x) {
            smem[tid] += smem[tid + stride];
        }
        __syncthreads();
    }
    
    // Write the result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = smem[0];
    }
}