#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixOnGPU2D(int *MatA, int *MatB, int *MatC, int nx, int ny)
{
    // Calculate the unique thread index across the grid
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Calculate the linear index of the matrix element
    unsigned int idx = iy * nx + ix;

    // Ensure threads work only on valid indices
    if (ix < nx && iy < ny) {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}