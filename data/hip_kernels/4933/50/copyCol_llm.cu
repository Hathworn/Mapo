#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyCol(float *out, float *in, const int nx, const int ny)
{
    // Compute global linear index to improve memory access pattern
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x + (blockDim.y * blockIdx.y + threadIdx.y) * nx;

    // Check bounds and copy column directly using linear index
    if (idx < nx * ny)
    {
        out[idx] = in[idx];
    }
}