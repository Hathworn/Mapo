#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceCompleteUnrollWarps8Sync(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n)
    {
        int sum = g_idata[idx] + g_idata[idx + blockDim.x] 
                + g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x] 
                + g_idata[idx + 4 * blockDim.x] + g_idata[idx + 5 * blockDim.x] 
                + g_idata[idx + 6 * blockDim.x] + g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = sum;
    }

    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];

    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];

    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];

    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];

    __syncthreads();

    // unrolling warp using loop
    if (tid < 32) 
    {
        #pragma unroll
        for (int offset = 32; offset > 0; offset /= 2)
        {
            idata[tid] += idata[tid + offset];
            __syncthreads();
        }
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}