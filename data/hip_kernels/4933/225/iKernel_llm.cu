#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iKernel(float *src, float *dst)
{
    // Use shared memory to reduce global memory accesses
    __shared__ float sharedSrc[BLOCK_DIM_X];
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int tIdx = threadIdx.x;
    
    // Load data into shared memory
    sharedSrc[tIdx] = src[idx];
    __syncthreads();
    
    // Perform computation using shared memory
    if (idx < SIZE) { // Boundary check to prevent out-of-bounds access
        dst[idx] = sharedSrc[tIdx] * 2.0f;
    }
}
```
