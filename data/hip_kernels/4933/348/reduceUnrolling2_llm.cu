#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling2(int *g_idata, int *g_odata, unsigned int n)
{
    // Set the thread id.
    unsigned int tid = threadIdx.x;
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x * 2;

    // Convert global data pointer to the local pointer of this block.
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;
    
    // Unrolling 2 data blocks.
    if (idx + blockDim.x < n)
    {
        g_idata[idx] += g_idata[idx + blockDim.x];
    }
    __syncthreads();

    // Boundary check.
    if (idx >= n) return;

    // In-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        // Synchronize within thread block.
        __syncthreads();
    }

    // Write result for this block to global memory.
    if (tid == 0)
    {
        g_odata[blockIdx.x] = idata[0];
    }
}