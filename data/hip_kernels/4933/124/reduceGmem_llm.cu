#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceGmem(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;
    
    // Boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    // Use loops instead of unrolling for better performance and scalability
    for (unsigned int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (tid < offset) {
            idata[tid] += idata[tid + offset];
        }
        __syncthreads(); // Ensure all threads have completed
    }

    // Write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}