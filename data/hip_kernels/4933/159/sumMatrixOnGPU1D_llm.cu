#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int gridSize = blockDim.x * gridDim.x; // Calculate total number of threads processing columns

    // Use stride to process elements beyond the current block
    for (int idx = ix; idx < nx * ny; idx += gridSize)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}