#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *out, float *in, const int nx, const int ny)
{
    // Use shared memory for coalesced memory access
    __shared__ float tile[BLOCK_SIZE][BLOCK_SIZE];

    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        // Load data from global to shared memory
        tile[threadIdx.y][threadIdx.x] = in[iy * nx + ix];
        __syncthreads();

        // Write data from shared to global memory
        out[iy * nx + ix] = tile[threadIdx.y][threadIdx.x];
    }
}