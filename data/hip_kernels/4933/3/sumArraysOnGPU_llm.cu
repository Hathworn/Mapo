#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds
    if (idx < N)
    {
        // Perform vector addition
        C[idx] = A[idx] + B[idx];
    }
}