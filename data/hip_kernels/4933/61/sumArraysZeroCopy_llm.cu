#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysZeroCopy(float *A, float *B, float *C, const int N)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}