#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceSmemOptimized(int *g_idata, int *g_odata, unsigned int n)
{
    __shared__ int smem[DIM];

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return;

    int *idata = g_idata + blockIdx.x * blockDim.x;
    smem[tid] = (tid < n) ? idata[tid] : 0; // Load with boundary check
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) { // Loop to reduce lines
        if (tid < s) smem[tid] += smem[tid + s];
        __syncthreads();
    }

    if (tid < 32) {
        volatile int *vsmem = smem; // Use volatile to prevent unwanted optimizations
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    if (tid == 0) g_odata[blockIdx.x] = smem[0]; // Store result
}