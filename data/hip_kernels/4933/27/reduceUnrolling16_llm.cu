#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling16(int *g_idata, int *g_odata, unsigned int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 16 + tid;
    
    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 16;
    
    // unrolling 16 with shared memory optimization
    __shared__ int sdata[256]; // use shared memory to reduce global reads
    int localSum = 0;

    if (idx + 15 * blockDim.x < n) {
        localSum = g_idata[idx] + g_idata[idx + blockDim.x] 
                 + g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x]
                 + g_idata[idx + 4 * blockDim.x] + g_idata[idx + 5 * blockDim.x]
                 + g_idata[idx + 6 * blockDim.x] + g_idata[idx + 7 * blockDim.x]
                 + g_idata[idx + 8 * blockDim.x] + g_idata[idx + 9 * blockDim.x]
                 + g_idata[idx + 10 * blockDim.x] + g_idata[idx + 11 * blockDim.x]
                 + g_idata[idx + 12 * blockDim.x] + g_idata[idx + 13 * blockDim.x]
                 + g_idata[idx + 14 * blockDim.x] + g_idata[idx + 15 * blockDim.x];
    }
    sdata[tid] = localSum;
    __syncthreads();

    // in-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads(); // synchronize within threadblock
    }
    
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}