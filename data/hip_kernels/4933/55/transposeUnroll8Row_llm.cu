#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll8Row(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 8 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    // Use shared memory to optimize memory access
    __shared__ float tile[8][8+1];

    if (ix + 7 * blockDim.x < nx && iy < ny)
    {
        #pragma unroll
        for (int i = 0; i < 8; i++)
        {
            tile[threadIdx.y][threadIdx.x + i * blockDim.x] = in[ti + i * blockDim.x];
        }
    }
    __syncthreads();

    ix = blockDim.y * blockIdx.y * 8 + threadIdx.x; // Transpose indexing
    iy = blockDim.x * blockIdx.x + threadIdx.y;

    if (ix < ny && iy + 7 * blockDim.y < nx)
    {
        #pragma unroll
        for (int i = 0; i < 8; i++)
        {
            out[to + i * blockDim.y] = tile[threadIdx.x][threadIdx.y + i];
        }
    }
}