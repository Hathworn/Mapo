#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeNaiveCol(int *in, int *out, const int nx, const int ny)
{
    // Calculate the flatten thread id from 2D block and thread index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < nx && idy < ny)
    {
        // More efficient memory access by using shared memory (eliminated)
        out[idy * nx + idx] = in[idx * ny + idy];
    }
}