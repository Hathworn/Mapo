#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmup(int *in, int *out, const int nx, const int ny)
{
    // Calculate thread id using block and thread indices
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Check if the thread is within matrix bounds
    if (ix < nx && iy < ny)
    {
        // Optimize matrix indexing using pre-calculated index
        unsigned int in_idx = iy + ix * ny;
        unsigned int out_idx = ix + iy * nx;
        out[out_idx] = in[in_idx];
    }
}