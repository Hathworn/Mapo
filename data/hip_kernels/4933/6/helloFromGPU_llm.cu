#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU(void)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index
    printf("Hello from GPU - global thread idx: %d\n", idx); // Use global index for efficient logging
}