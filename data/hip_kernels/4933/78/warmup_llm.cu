#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *A, float *B, float *C, const int N)
{
    // Use shared memory for better memory access performance
    extern __shared__ float shared[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        shared[threadIdx.x] = A[i] + B[i];
        __syncthreads(); // Ensure all threads have written to shared memory
        C[i] = shared[threadIdx.x];
    }
}