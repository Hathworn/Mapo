#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll4Col(int *in, int *out, const int nx, const int ny)
{
    // Set thread id.
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x * 4;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Calculate indices for accessing rows and columns
    unsigned int ti = iy * nx + ix; // Access in rows.
    unsigned int to = ix * ny + iy; // Access in cols.

    // Unroll the loop to optimize memory access
    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        // Use temporary variables to reduce repetitive calculations
        unsigned int step = ny * blockDim.x;
        out[ti]              = in[to];
        out[ti + blockDim.x] = in[to + step];
        out[ti + 2 * blockDim.x] = in[to + 2 * step];
        out[ti + 3 * blockDim.x] = in[to + 3 * step];
    }
}