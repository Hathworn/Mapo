#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void device_api_kernel(hiprandState *states, float *out, int N)
{
int i;
int tid = blockIdx.x * blockDim.x + threadIdx.x;
int nthreads = gridDim.x * blockDim.x;
hiprandState *state = states + tid;

hiprand_init(9384, tid, 0, state);

for (i = tid; i < N; i += nthreads)
{
float rand = hiprand_uniform(state);
rand = rand * 2;
out[i] = rand;
}
}