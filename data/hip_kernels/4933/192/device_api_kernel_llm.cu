#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_api_kernel(hiprandState *states, float *out, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;

    // Initialize the random state for the current thread
    hiprandState localState;
    hiprand_init(9384, tid, 0, &localState);

    // Use shared state instead of recomputing state pointer
    for (int i = tid; i < N; i += nthreads)
    {
        // Generate random number and store result
        float rand = hiprand_uniform(&localState);
        rand = rand * 2.0f;
        out[i] = rand;
    }
}