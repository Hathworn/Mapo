#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iKernel(float *src, float *dst, int n)
{
    // Calculate global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check index is within the bounds
    if (idx < n) {
        dst[idx] = src[idx] * 2.0f;
    }
}
```
