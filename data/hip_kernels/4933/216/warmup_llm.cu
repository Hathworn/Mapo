#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Use shared memory for frequently accessed data
    extern __shared__ float sharedMem[];

    if (k < n) {
        sharedMem[threadIdx.x] = A[k] + B[k];
        __syncthreads(); // Ensure all writes to shared memory are complete
        C[i] = sharedMem[threadIdx.x]; // Write result to global memory
    }
}