#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void checkIndex(void)
{
    // Compute once, reuse in prints for efficiency
    int thx = threadIdx.x, thy = threadIdx.y, thz = threadIdx.z;
    int blx = blockIdx.x, bly = blockIdx.y, blz = blockIdx.z;

    // Consolidated prints to one line for conciseness and less overhead
    printf("threadIdx:(%d, %d, %d), blockIdx:(%d, %d, %d)\n", thx, thy, thz, blx, bly, blz);
    printf("blockDim:(%d, %d, %d), gridDim:(%d, %d, %d)\n", blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}