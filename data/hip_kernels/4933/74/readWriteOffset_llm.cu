#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readWriteOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate global thread ID with offset
    unsigned int k = blockIdx.x * blockDim.x + threadIdx.x + offset;

    // Check within bounds and compute the result
    if (k < n) {
        C[k] = A[k] + B[k];
    }
}