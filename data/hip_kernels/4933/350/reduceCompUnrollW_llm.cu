#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceCompUnrollW(int *g_idata, int *g_odata, unsigned int n)
{
    // Set the thread id
    unsigned int tid = threadIdx.x;
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x * 8;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // Unrolling 8 data blocks
    int sum = 0;
    #pragma unroll
    for (int i = 0; i < 8; i++)
    {
        if (idx + i * blockDim.x < n)
        {
            sum += g_idata[idx + i * blockDim.x];
        }
    }
    idata[tid] = sum;
    __syncthreads();

    // In-place reduction with complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();
    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();
    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();
    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();

    // Unrolling warp reduction
    if (tid < 32)
    {
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0)
    {
        g_odata[blockIdx.x] = idata[0];
    }
}