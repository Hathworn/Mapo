#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmemDyn(int *g_idata, int *g_odata, unsigned int n)
{
    extern __shared__ int smem[];

    // set thread ID
    unsigned int tid = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // set to smem by each thread
    smem[tid] = (tid < n) ? idata[tid] : 0;  // Ensure out-of-bounds threads don't access g_idata
    __syncthreads();

    // in-place reduction using loop unrolling
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            smem[tid] += smem[tid + s];
        }
        __syncthreads();
    }

    // unrolling warp, no __syncthreads() needed within a warp
    if (tid < 32) {
        volatile int *vsmem = smem;  // Ensure compiler doesn't reorder memory operations
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}