#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Compute the global thread index for 1D grid
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Use a single loop to iterate over both dimensions
    if (idx < nx * ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}