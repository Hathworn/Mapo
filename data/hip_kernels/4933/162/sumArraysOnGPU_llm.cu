#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // Use blockIdx and blockDim to handle larger arrays
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread processes within bounds
    if (i < N) C[i] = A[i] + B[i];
}