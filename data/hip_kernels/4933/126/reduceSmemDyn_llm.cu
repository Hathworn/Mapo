#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmemDyn(int *g_idata, int *g_odata, unsigned int n)
{
    extern __shared__ int smem[];

    // set thread ID
    unsigned int tid = threadIdx.x;

    // Improve memory coalescing by accessing contiguous memory
    int index = blockIdx.x * blockDim.x + tid;
    if (index < n) {
        smem[tid] = g_idata[index];
    } else {
        smem[tid] = 0; // Initialize out-of-bound threads to 0
    }
    __syncthreads();

    // in-place reduction in shared memory with loop unrolling
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            smem[tid] += smem[tid + s];
        }
        __syncthreads();
    }

    // Unrolling last warp without __syncthreads
    if (tid < 32)
    {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}