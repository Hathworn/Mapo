#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU(void)
{
    // Use one thread per block to reduce redundant printing
    if (threadIdx.x == 0 && blockIdx.x == 0) 
    {
        printf("Hello from GPU.\n");
    }
}