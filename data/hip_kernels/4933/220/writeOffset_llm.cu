#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate global index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Check bounds and perform addition
    // Use if statement within a single line for better performance
    if (k < n) C[k] = A[i] + B[i];
}