#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighbored (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    // in-place reduction in shared memory for improved performance
    extern __shared__ int sdata[];
    sdata[tid] = idata[tid];
    __syncthreads();

    // perform reduction
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
    {
        unsigned int index = 2 * stride * tid;
        if (index < blockDim.x)
        {
            sdata[index] += sdata[index + stride];
        }
        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}