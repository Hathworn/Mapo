#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1d(float *in, float *out, int N)
{
    // shared memory
    __shared__ float smem[BDIM + 2 * RADIUS];

    // index to global memory
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    while (idx < N)
    {
        // index to shared memory for stencil calculatioin
        int sidx = threadIdx.x + RADIUS;

        // Read data from global memory into shared memory
        if (idx < N) {
            smem[sidx] = in[idx];
            // read halo part to shared memory
            if (threadIdx.x < RADIUS) {
                smem[sidx - RADIUS] = (idx >= RADIUS) ? in[idx - RADIUS] : 0.0f; // Check boundary
                smem[sidx + BDIM] = ((idx + BDIM) < N) ? in[idx + BDIM] : 0.0f; // Check boundary
            }
        }

        // Synchronize (ensure all the data is available)
        __syncthreads();

        // Apply the stencil
        float tmp = 0.0f;

        // Unroll the loop with pragma
        #pragma unroll
        for (int i = 1; i <= RADIUS; i++)
        {
            tmp += coef[i] * (smem[sidx + i] - smem[sidx - i]);
        }

        // Store the result
        if (idx < N) {
            out[idx] = tmp;
        }

        idx += gridDim.x * blockDim.x;
    }
}