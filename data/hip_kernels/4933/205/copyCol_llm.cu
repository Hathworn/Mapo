#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyCol(float *out, float *in, const int nx, const int ny)
{
    // Use flattenId to eliminate index calculations in the loop
    unsigned int flattenId = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    
    // Loop over columns to leverage memory access efficiency
    for (unsigned int ix = flattenId; ix < nx; ix += stride) {
        for (unsigned int iy = threadIdx.y; iy < ny; iy += blockDim.y) {
            out[ix * ny + iy] = in[ix * ny + iy];
        }
    }
}