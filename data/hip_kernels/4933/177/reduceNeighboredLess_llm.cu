#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighboredLess (int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if(idx >= n) return;

    // Perform in-place reduction in shared memory for better performance
    extern __shared__ int sdata[];
    sdata[tid] = idata[tid];
    __syncthreads();

    // In-place reduction in shared memory
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
    {
        int index = 2 * stride * tid;

        if (index < blockDim.x)
        {
            sdata[index] += sdata[index + stride];
        }
        // Synchronize within threadblock
        __syncthreads();
    }

    // Write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}