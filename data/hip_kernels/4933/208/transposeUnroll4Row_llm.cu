#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll4Row(float *out, float *in, const int nx, const int ny)
{
    // Optimize by using registers and reducing redundant calculations
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        unsigned int ti = iy * nx + ix; // Compute once
        unsigned int to = ix * ny + iy; // Compute once

        float inVal0 = in[ti];
        float inVal1 = in[ti + blockDim.x];
        float inVal2 = in[ti + 2 * blockDim.x];
        float inVal3 = in[ti + 3 * blockDim.x];

        out[to]                   = inVal0;
        out[to + ny * blockDim.x]   = inVal1;
        out[to + ny * 2 * blockDim.x] = inVal2;
        out[to + ny * 3 * blockDim.x] = inVal3;
    }
}