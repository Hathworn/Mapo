#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index is within bounds and compute result once
    if (idx < N)
    {
        C[idx] = A[idx] + B[idx];
    }
}