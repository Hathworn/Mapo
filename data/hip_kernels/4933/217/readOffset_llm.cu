#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffset(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Combine index computation and boundary check
    if (i + offset < n) {
        // Precompute the offset index
        unsigned int k = i + offset;
        C[i] = A[k] + B[k];
    }
}