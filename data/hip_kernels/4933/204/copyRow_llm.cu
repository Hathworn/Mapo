#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyRow(float *out, float *in, const int nx, const int ny)
{
    // Calculate flattened thread index
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x + 
                       (threadIdx.y + blockIdx.y * blockDim.y) * nx;

    // Ensure index is within bounds and perform copy
    if (idx < nx * ny)
    {
        out[idx] = in[idx];
    }
}