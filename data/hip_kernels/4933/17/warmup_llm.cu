#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(int *out, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use if-else to branch only when necessary
    if (tid < N) {
        out[tid] = 0;
    }
}