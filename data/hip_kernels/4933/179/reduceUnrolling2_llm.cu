#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling2(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID and calculate index for unrolling
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + tid;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // Unrolling 2 operations
    int temp = 0;
    if (idx + blockDim.x < n) temp = g_idata[idx + blockDim.x];
    idata[tid] = g_idata[idx] + temp;

    __syncthreads();

    // In-place reduction in shared memory using tid
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}