#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_2dfd_last(float *g_u1, float *g_u2, const int nx, const int iStart, const int iEnd)
{
    unsigned int ix  = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stx = threadIdx.x + NPAD;
    unsigned int idx  = ix + iStart * nx;

    // Use shared memory to reduce global memory access
    __shared__ float tile[BDIMX + NPAD2];

    const float alpha = 0.12f;
    float yval[9];

    // Initialize yval with values from g_u2
    for (int i = 0; i < 8; i++)
        yval[i] = g_u2[idx + (i - 4) * nx];
    
    int iskip = NPAD * nx;

    #pragma unroll 9
    for (int iy = iStart; iy < iEnd; iy++)
    {
        yval[8] = g_u2[idx + iskip];

        // Load edges of tile in shared mem for thread synchronization
        if (threadIdx.x < NPAD)
        {
            tile[threadIdx.x] = g_u2[idx - NPAD];
            tile[stx + BDIMX] = g_u2[idx + BDIMX];
        }

        tile[stx] = yval[4];
        __syncthreads();

        if ((ix >= NPAD) && (ix < nx - NPAD))
        {
            float tmp = coef[0] * tile[stx] * 2.0f;

            #pragma unroll
            for (int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (tile[stx - d] + tile[stx + d]);
            }

            #pragma unroll
            for (int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (yval[4 - d] + yval[4 + d]);
            }

            // Compute time dimension and update global memory
            g_u1[idx] = yval[4] + yval[4] - g_u1[idx] + alpha * tmp;
        }

        #pragma unroll 8
        for (int i = 0; i < 8; i++)
        {
            yval[i] = yval[i + 1];
        }

        idx += nx;
        __syncthreads();
    }
}