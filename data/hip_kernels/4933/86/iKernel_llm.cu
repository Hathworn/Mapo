#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iKernel(float *A, float *B, float *C, const int N)
{
    // Use a loop to ensure all elements are processed even if N is not a multiple of blockDim.x
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    {
        C[i] = A[i] + B[i];
    }
}