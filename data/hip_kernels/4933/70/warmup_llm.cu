#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *A, float *B, float *C, const int N)
{
    // Improved memory access by using shared memory
    extern __shared__ float shared_memory[];
    float* sA = shared_memory;
    float* sB = shared_memory + blockDim.x;

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data from global to shared memory
    if (i < N) {
        sA[threadIdx.x] = A[i];
        sB[threadIdx.x] = B[i];
    }
    __syncthreads();

    // Ensure we don't access out of bounds
    if (i < N) C[i] = sA[threadIdx.x] + sB[threadIdx.x];
}