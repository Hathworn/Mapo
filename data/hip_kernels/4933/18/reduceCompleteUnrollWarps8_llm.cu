#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceCompleteUnrollWarps8(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // Unrolling 8
    if (idx + 7 * blockDim.x < n)
    {
        int sum = g_idata[idx] + g_idata[idx + blockDim.x] + 
                  g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x] + 
                  g_idata[idx + 4 * blockDim.x] + g_idata[idx + 5 * blockDim.x] + 
                  g_idata[idx + 6 * blockDim.x] + g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = sum;
    }

    __syncthreads();

    // In-place reduction and complete unroll with loop
    #pragma unroll
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tid < offset) idata[tid] += idata[tid + offset];
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}