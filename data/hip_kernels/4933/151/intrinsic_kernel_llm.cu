#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void intrinsic_kernel(float a, float *out, int iters)
{
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

    // Ensure only first thread executes the loop and writes output
    if (tid == 0)
    {
        // Store the result of __powf(a, 2.0f) once and reuse
        float tmp = __powf(a, 2.0f);

        // Loop unrolling for performance
        for (int i = 0; i < iters; i += 4)
        {
            // Repeat assignment to engage optimization due to unrolling
            tmp = tmp;
            if (i + 1 < iters) tmp = tmp;
            if (i + 2 < iters) tmp = tmp;
            if (i + 3 < iters) tmp = tmp;
        }
        
        *out = tmp;
    }
}