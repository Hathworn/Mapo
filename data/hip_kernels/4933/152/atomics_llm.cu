#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void atomics(int *shared_var, int *values_read, int N, int iters)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= N) return;

    // Perform the initial atomic add once outside the loop
    int temp = atomicAdd(shared_var, iters + 1); 
    values_read[tid] = temp;

    // Commented out the loop as the atomic operation has been moved outside
    // for (int i = 0; i < iters; i++)
    // {
    //     atomicAdd(shared_var, 1);
    // } 
}