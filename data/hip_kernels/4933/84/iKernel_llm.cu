#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iKernel(float *src, float *dst)
{
    // Use shared memory for faster access if needed
    extern __shared__ float sdata[];

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    sdata[threadIdx.x] = src[idx];

    // Wait for all threads to finish loading
    __syncthreads();

    // Compute using shared memory
    dst[idx] = sdata[threadIdx.x] * 2.0f;
}