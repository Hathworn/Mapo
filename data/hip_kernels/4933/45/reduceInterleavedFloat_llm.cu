#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceInterleavedFloat (float *g_idata, float *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if(idx >= n) return;

    // In-place reduction within this block
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads(); // Ensure all threads have completed this stride before moving to the next
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}