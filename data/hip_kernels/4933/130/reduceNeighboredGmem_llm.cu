#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighboredGmem(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n) return;

    // in-place reduction in global memory using shared memory to reduce global memory access
    extern __shared__ int sdata[];
    sdata[tid] = g_idata[idx];
    __syncthreads();

    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if (tid % (2 * stride) == 0)
        {
            sdata[tid] += sdata[tid + stride];
        }
        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}