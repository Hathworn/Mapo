#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void warmup(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int index = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        out[index] = in[index];
    }
}