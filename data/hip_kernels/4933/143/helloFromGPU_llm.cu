#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory to optimize thread usage
__global__ void helloFromGPU()
{
    // Calculate unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Only the first thread in each block will print to reduce overhead
    if (idx == 0)
    {
        printf("Hello World from GPU!\n");
    }
}