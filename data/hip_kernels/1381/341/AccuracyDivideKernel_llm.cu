#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Each thread gets an index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Only the first thread performs the division to avoid race conditions
    if (idx == 0) {
        *accuracy /= N;
    }
}