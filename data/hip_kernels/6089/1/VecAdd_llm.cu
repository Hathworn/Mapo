#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure that the thread accesses valid memory
    if (tid < N) {
        C[tid] = A[tid] + B[tid];
    }
}