#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void sumArraysOnGPU(float *A, float *B, float *C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < n) // Ensure thread does not access out-of-bounds memory
    {
        C[i] = A[i] + B[i]; // Perform the sum
    }
}
```
