#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 256

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

// Optimized kernel to utilize block and grid dimensions
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure threads don't access out-of-bound memory
    if (i < size)
    {
        c[i] = a[i] + b[i];
    }
}