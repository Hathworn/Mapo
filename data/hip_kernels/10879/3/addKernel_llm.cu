#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Utilize block-level parallelism for potentially larger arrays
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if (i < arraySize) // Ensure index is within bounds
    {
        c[i] = a[i] + b[i]; // Perform addition
    }
}
```
