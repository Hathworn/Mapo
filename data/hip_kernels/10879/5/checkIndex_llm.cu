#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void checkIndex(void) {
    // Calculate global thread index in 1D
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) { // Ensure only one thread performs printf
        printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) "
               "gridDim:(%d, %d, %d)\n",
               threadIdx.x, threadIdx.y, threadIdx.z,
               blockIdx.x, blockIdx.y, blockIdx.z,
               blockDim.x, blockDim.y, blockDim.z,
               gridDim.x, gridDim.y, gridDim.z);
    }
}