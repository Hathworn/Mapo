#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(int *MatA, int *MatB, int *MatC, int nx, int ny)
{
    // Use shared memory for better memory access patterns
    __shared__ int sharedA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int sharedB[BLOCK_SIZE][BLOCK_SIZE];

    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    
    // Load data into shared memory
    if (ix < nx && iy < ny) {
        sharedA[threadIdx.y][threadIdx.x] = MatA[idx];
        sharedB[threadIdx.y][threadIdx.x] = MatB[idx];
    }
    __syncthreads();

    // Perform addition
    if (ix < nx && iy < ny)
        MatC[idx] = sharedA[threadIdx.y][threadIdx.x] + sharedB[threadIdx.y][threadIdx.x];
}