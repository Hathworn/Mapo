#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_check(int threads, uint64_t *data, uint32_t *results, uint64_t target)
{
    __shared__ uint32_t tmp[512/32];

    int thread = (blockDim.x * blockIdx.x + threadIdx.x);

    // Initialize shared memory array using all threads in warp
    int warpIndex = threadIdx.x / 32;
    int laneIndex = threadIdx.x % 32;
    if (laneIndex == 0) 
        tmp[warpIndex] = 0;

    __syncthreads();

    if (thread < threads)
    {
        uint64_t highword = data[threads*3 + thread];
        if(highword < target){
            atomicOr(&tmp[warpIndex], 1 << laneIndex); // Coalesced atomic operation
        }

        __syncthreads();
        if (laneIndex == 0) // Store final result using a single thread per warp
            results[blockIdx.x*(4096/32) + warpIndex] = tmp[warpIndex];
    }
}