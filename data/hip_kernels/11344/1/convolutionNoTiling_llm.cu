#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define _USE_MATH_DEFINES

static void CheckCudaErrorAux(const char *, unsigned, const char *,
hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

__global__ void convolutionNoTiling(float *I, float *P, int channels, int width, int height) {

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int depth = threadIdx.z;

    if (col < width && row < height && depth < channels) {

        float pValue = 0.0f;

        int startRow = row - maskRowsRadius;
        int startCol = col - maskColumnsRadius;

        // Optimize loop by precomputing boundaries for ghost elements
        int rowLowerBound = max(0, startRow);
        int rowUpperBound = min(height, startRow + maskRows);
        int colLowerBound = max(0, startCol);
        int colUpperBound = min(width, startCol + maskColumns);

        for (int i = rowLowerBound; i < rowUpperBound; i++) {
            for (int j = colLowerBound; j < colUpperBound; j++) {
                float iValue = I[(i * width + j) * channels + depth];
                pValue += iValue * deviceMaskData[(i - startRow) * maskColumns + (j - startCol)];
            }
        }

        // Save result from register to global
        P[(row * width + col) * channels + depth] = pValue;
    }
}