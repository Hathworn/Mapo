#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InitArrays(float *ip, float *op, float *fp, int *kp, int ncols)
{
    int blockOffset = blockIdx.x * ncols;   // Calculate the offset of the row
    int colsPerThread = ncols / blockDim.x; // The number of cols per thread
    int rowStartPos = threadIdx.x * colsPerThread; // Calculate offset into the row

    float* ippos = ip + blockOffset + rowStartPos;
    float* fppos = fp + blockOffset + rowStartPos;
    float* oppos = op + blockOffset + rowStartPos;
    int* kppos = kp + blockOffset + rowStartPos;

    // Initialize arrays
    for (int i = 0; i < colsPerThread; i++) {
        fppos[i] = NOTSETLOC;
        ippos[i] = 50;
        oppos[i] = 50;
        kppos[i] = 1;
    }

    // Set boundary conditions
    if (rowStartPos == 0 || blockOffset == 0) {
        for (int i = 0; i < colsPerThread; i++) {
            fppos[i] = SETLOC;
            ippos[i] = 0;
            oppos[i] = 0;
            kppos[i] = 0;
        }
    }

    if (rowStartPos + colsPerThread >= ncols || blockOffset == ncols - 1) {
        for (int i = 0; i < colsPerThread; i++) {
            fppos[i] = SETLOC;
            ippos[i] = 100;
            oppos[i] = 100;
            kppos[i] = 0;
        }
    }

    // Special cases
    if (blockOffset == 400 && rowStartPos < 330) {
        int end = min(colsPerThread, 330 - rowStartPos);
        for (int i = 0; i < end; i++) {
            fppos[i] = SETLOC;
            ippos[i] = 100;
            oppos[i] = 100;
            kppos[i] = 0;
        }
    } else if (blockOffset == 200 && rowStartPos <= 500 && rowStartPos + colsPerThread >= 500) {
        int i = 500 - rowStartPos;
        fppos[i] = SETLOC;
        ippos[i] = 100;
        oppos[i] = 100;
        kppos[i] = 0;
    }
}