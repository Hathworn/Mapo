#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iReduceSum(int *idata, int *odata, unsigned int ncols) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + tid;
    extern __shared__ int sdata[];

    int myPart = 0;
    int colsPerThread = ncols / (gridDim.x * blockDim.x);

    // Load input elements into shared memory and accumulate
    for(int i = 0; i < colsPerThread; i++) {
        myPart += idata[idx + i * blockDim.x];
    }
    sdata[tid] = myPart;
    __syncthreads();

    // Perform reduction in shared memory
    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if(tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if(tid == 0) {
        odata[blockIdx.x] = sdata[0];
    }
}