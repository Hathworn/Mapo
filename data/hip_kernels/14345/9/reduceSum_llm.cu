#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceSum(int *idata, int *odata, unsigned int ncols)
{
    // Use shared memory for partial sums
    extern __shared__ int sharedData[];
    int *sdata = sharedData;

    int tid = threadIdx.x;
    int blockOffset = blockIdx.x * ncols;
    int rowStartPos = tid * (ncols / blockDim.x);
    int colsPerThread = ncols / blockDim.x;

    int *mypart = idata + blockOffset + rowStartPos;

    // Load data into shared memory
    sdata[tid] = mypart[0];
    for (int i = 1; i < colsPerThread; i++) {
        sdata[tid] += mypart[i];
    }
    __syncthreads(); // Ensure all loads are done

    // Parallel reduction within a block
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // Ensure all adds are done
    }

    // Save the final result from each block
    if (tid == 0) {
        odata[blockIdx.x] = sdata[0];
    }
}