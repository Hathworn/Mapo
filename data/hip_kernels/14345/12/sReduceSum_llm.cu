#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sReduceSum(int *idata, int *odata, unsigned int ncols) {
    unsigned int tid = threadIdx.x;
    unsigned int colsPerThread = ncols / blockDim.x;
    unsigned int blockOffset = blockIdx.x * ncols + tid * colsPerThread;
    
    // Use shared memory initialization and avoid redundant memory assignments
    extern __shared__ int sdata[];
    int myPart = 0;

    // Unroll loops for better performance
    for (unsigned int i = 0; i < colsPerThread; i++) {
        myPart += idata[blockOffset + i];
    }
    sdata[tid] = myPart;
    __syncthreads();

    // Optimize reduction with loop unrolling
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result to global memory
    if (tid == 0) {
        odata[blockIdx.x] = sdata[0];
    }
}