#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iReduceSingle(int *idata, int *single, int ncols) {
    unsigned int tid = threadIdx.x;
    extern __shared__ int sdata[];

    int colsPerThread = ncols / blockDim.x;
    int myPart = 0;

    // Optimize the initial reduction within the block
    for (int i = tid; i < ncols; i += blockDim.x) {
        myPart += idata[i];
    }
    sdata[tid] = myPart;
    __syncthreads();

    // Unroll the loop for intra-block reduction to achieve better parallel performance
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write the block's result back to global memory
    if (tid == 0) *single = sdata[0];
}