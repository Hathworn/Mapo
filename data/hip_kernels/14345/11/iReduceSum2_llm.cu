#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iReduceSum2(int *idata, int *odata, unsigned int ncols) {
    // Calculate thread ID and block offset
    unsigned int tid = threadIdx.x;
    unsigned int startPos = blockDim.x * blockIdx.x;
    extern __shared__ int sdata[];

    // Calculate number of elements per thread and block offset
    int colsPerThread = (ncols + blockDim.x - 1) / blockDim.x; // Use ceiling to cover all elements
    int blockOffset = tid * colsPerThread;

    // Initialize partial sum for each thread
    int myPart = 0;
    for(int i = 0; i < colsPerThread && blockOffset + startPos + i < ncols; i++) {
        myPart += idata[blockOffset + startPos + i];
    }

    // Each thread writes its partial result to shared memory
    sdata[tid] = myPart;
    __syncthreads();

    // Perform parallel reduction within block
    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result of block reduction to odata
    if(tid == 0) odata[blockIdx.x] = sdata[0];
}
```
