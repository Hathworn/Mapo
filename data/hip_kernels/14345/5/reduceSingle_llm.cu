#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceSingle(int *idata, int *single, int nrows)
{
    extern __shared__ int parts[];

    // Initialize shared memory
    int sum = 0;
    for (int i = threadIdx.x; i < nrows; i += blockDim.x) {
        sum += idata[i];
    }
    parts[threadIdx.x] = sum;
    __syncthreads();

    // Optimize sum reduction using loop unrolling
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (threadIdx.x < stride) {
            parts[threadIdx.x] += parts[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Unrolling the last warp
    if (threadIdx.x < 32) {
        volatile int *vparts = parts;
        vparts[threadIdx.x] += vparts[threadIdx.x + 32];
        vparts[threadIdx.x] += vparts[threadIdx.x + 16];
        vparts[threadIdx.x] += vparts[threadIdx.x + 8];
        vparts[threadIdx.x] += vparts[threadIdx.x + 4];
        vparts[threadIdx.x] += vparts[threadIdx.x + 2];
        vparts[threadIdx.x] += vparts[threadIdx.x + 1];
    }

    // Only the first thread writes the result
    if (threadIdx.x == 0) {
        *single = parts[0];
    }
}