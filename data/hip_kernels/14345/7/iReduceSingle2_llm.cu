#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iReduceSingle2(int *idata, int *single, unsigned int ncols) {
    unsigned int tid = threadIdx.x;
    extern __shared__ int sdata[];
    unsigned int startPos = tid * (ncols / blockDim.x);  // Updated to ensure evenly distributed work
    int myPart = 0;
    
    // Use loop unrolling for improved memory throughput
    int step = ncols / blockDim.x;
    for (int i = startPos; i < startPos + step; i += 4) {
        if (i < ncols) myPart += idata[i];
        if (i + 1 < ncols) myPart += idata[i + 1];
        if (i + 2 < ncols) myPart += idata[i + 2];
        if (i + 3 < ncols) myPart += idata[i + 3];
    }
    
    sdata[tid] = myPart;
    __syncthreads();

    // Optimized reduction with warp divergence minimization
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) *single = sdata[0];
}
```
