#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sReduceSingle(int *idata, int *single, unsigned int ncols) {
    unsigned int tid = threadIdx.x;
    extern __shared__ int sdata[];
    unsigned int startPos = blockDim.x + tid;
    int colsPerThread = ncols / blockDim.x;
    int myPart = 0;

    // Loop unrolling for better performance
    for (int i = 0; i < colsPerThread; i += 4) {
        myPart += idata[startPos + i];
        if (i+1 < colsPerThread) myPart += idata[startPos + i+1];
        if (i+2 < colsPerThread) myPart += idata[startPos + i+2];
        if (i+3 < colsPerThread) myPart += idata[startPos + i+3];
    }
    
    sdata[tid] = myPart;
    __syncthreads();

    // Optimized reduction loop using warp shuffling
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) *single = sdata[0];
}