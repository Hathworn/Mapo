#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    #pragma unroll
    for (int i = 0; i < 13; i++)
        hist[i * 32 + tx] = 0.0f;
    __syncthreads();

    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    if (tx < 15)
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));
    
    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;
    int px = xp & 15;
    int x = tx - px;

    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = xp + x;
        int yi = yp + y;
        xi = max(0, min(xi, w - 1));
        yi = max(0, min(yi, h - 1));
        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    #pragma unroll
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = int(16.0f * atan2f(dy, dx) / 3.1416f + 16.5f) % 32;
            float grad = sqrtf(dx * dx + dy * dy);
            hist[32 * (x - 1) + bin] += grad * gauss[x] * gauss[y];
        }
    }
    __syncthreads();

    // Reduce histogram values for better performance
    for (int y = 0; y < 4; y++) 
        hist[y * 32 + tx] += hist[(y + 4) * 32 + tx] + hist[(y + 8) * 32 + tx];
    __syncthreads();
    
    for (int y = 0; y < 2; y++)
        hist[y * 32 + tx] += hist[(y + 2) * 32 + tx];
    
    __syncthreads();
    
    hist[tx] += hist[32 + tx];
    __syncthreads();

    // Calculate weighted histogram
    if (tx < 32) {
        int idx1 = (tx + 1) & 31;
        int idx2 = (tx + 31) & 31;
        float v = hist[32 + tx];
        hist[tx] = (v > hist[32 + idx1] && v >= hist[32 + idx2]) ? v : 0.0f;
    }
    __syncthreads();

    // Find peaks and second peak
    if (tx == 0) {
        float maxval1 = 0.0f, maxval2 = 0.0f;
        int i1 = -1, i2 = -1;

        #pragma unroll
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }

        float peakOffset1 = 0.5f * (hist[32 + ((i1 + 1) & 31)] - hist[32 + ((i1 + 31) & 31)]) / (2.0f * maxval1 - hist[32 + ((i1 + 1) & 31)] - hist[32 + ((i1 + 31) & 31)]);
        d_Orient[bx] = 11.25f * ((peakOffset1 < 0.0f) ? peakOffset1 + 32.0f : peakOffset1);

        if (maxval2 < 0.8f * maxval1)
            i2 = -1;

        if (i2 >= 0) {
            float peakOffset2 = 0.5f * (hist[32 + ((i2 + 1) & 31)] - hist[32 + ((i2 + 31) & 31)]) / (2.0f * maxval2 - hist[32 + ((i2 + 1) & 31)] - hist[32 + ((i2 + 31) & 31)]);
            d_Orient[bx + maxPts] = 11.25f * ((peakOffset2 < 0.0f) ? peakOffset2 + 32.0f : peakOffset2);
        } else {
            d_Orient[bx + maxPts] = i2;
        }
    }
}