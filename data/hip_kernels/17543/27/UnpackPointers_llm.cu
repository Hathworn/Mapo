#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;

    // Calculate stride for y loop to optimize memory usage
    const int yStride = w * 32;
    const int xStride = 16;

    // Optimized loop to reduce number of iterations
    for (int y = 0; y < h; y += 32) {
        for (int x = 0; x < w; x += xStride) {
            unsigned int val = minmax[(y / 32) * w + x + tx];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[numPts * xStride + tx] = (y + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}