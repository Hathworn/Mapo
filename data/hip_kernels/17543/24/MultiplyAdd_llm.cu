#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use dim3 for block size initialization for better readability and flexibility
__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Calculate the global index in a more straightforward way
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;
    
    // Check boundary conditions before accessing data
    if (x < width && y < height) {
        // Optimize by removing __syncthreads(), not needed here
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
}