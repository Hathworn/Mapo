#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Calculate unique thread index considering block and thread dimensions
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;

    // Boundary check to ensure threads only operate within valid data range
    if (x < width && y < height) {
        // Perform subtraction only if within boundaries
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }

    // __syncthreads() is unnecessary here as no shared memory is involved
}