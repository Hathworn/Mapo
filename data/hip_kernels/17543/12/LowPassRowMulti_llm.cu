#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LowPassRowMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    __shared__ float data[CONVROW_W + 2*RADIUS];
    const int tx = threadIdx.x;
    const int block = blockIdx.x / (NUM_SCALES + 3);
    const int scale = blockIdx.x - (NUM_SCALES + 3) * block;
    const int xout = block * CONVROW_W + tx;
    const int loadPos = xout - RADIUS;
    const int yptr = blockIdx.y * pitch;
    const int writePos = yptr + height * pitch * scale + xout;
    float *kernel = d_Kernel + scale * 16;

    // Load data with boundary check using min-max to avoid branching
    data[tx] = d_Data[yptr + max(0, min(loadPos, width - 1))];
    __syncthreads();

    // Process only valid threads
    if (xout < width && tx < CONVROW_W) {
        d_Result[writePos] =
            (data[tx + 0] + data[tx + 8]) * kernel[0] +
            (data[tx + 1] + data[tx + 7]) * kernel[1] +
            (data[tx + 2] + data[tx + 6]) * kernel[2] +
            (data[tx + 3] + data[tx + 5]) * kernel[3] +
            data[tx + 4] * kernel[4];
    }
    __syncthreads();
}