#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    const int gridSizeX = gridDim.x * blockDim.x;
    int numPts = 0;

    for (int y = bx; y < h / 32; y += gridSizeX / 16) {  // Efficient index calculation for y
        for (int x = tx; x < w; x += blockDim.x) {  // Efficient index calculation for x
            unsigned int val = minmax[y * w + x];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    // Unroll the loop for performance improvement
                    #pragma unroll
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[gridSizeX * numPts + x] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}