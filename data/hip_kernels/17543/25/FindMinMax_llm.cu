#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    const int b = blockDim.x;
    int p = y * width + x;

    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        minvals[tx] = FLT_MAX;
        maxvals[tx] = -FLT_MAX;
    }

    for (int ty = 1; ty < 16; ty++) {
        p += width;
        if (x < width) { // corrected from tx to x
            float val = d_Data[p];
            if (val < minvals[tx]) minvals[tx] = val;
            if (val > maxvals[tx]) maxvals[tx] = val;
        }
    }
    
    __syncthreads();

    for (int d = b/2; d > 0; d >>= 1) { // Reduce using binary tree structure
        if (tx < d) {
            if (minvals[tx + d] < minvals[tx]) minvals[tx] = minvals[tx + d];
            if (maxvals[tx + d] > maxvals[tx]) maxvals[tx] = maxvals[tx + d];
        }
        __syncthreads();
    }

    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}