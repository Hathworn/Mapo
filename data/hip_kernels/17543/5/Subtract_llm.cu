#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    // Calculate global thread coordinates
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate linear index in the data arrays
    int p = y * pitch + x;

    // Boundary check to ensure threads operate within matrix dimensions
    if (x < width && y < height) {
        // Perform subtraction operation
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }

    // Remove unnecessary __syncthreads
}