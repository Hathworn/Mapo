#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mkRender(float *fb, int max_x, int max_y) {
    // Compute pixel position using ThreadId and BlockId
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    // Skip computations if pixel position is outside FB size
    if (i >= max_x || j >= max_y) return;

    // Calculate FB pixel value
    int pixel_index = (j * max_x + i) * 3;
    fb[pixel_index] = float(i) / max_x;
    fb[pixel_index + 1] = float(j) / max_y;
    fb[pixel_index + 2] = 0.2f;
}