#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void partialScan(unsigned int *d_in, unsigned int *d_out, unsigned int *d_total, size_t n)
{
    __shared__ unsigned int temp[BLOCK_WIDTH];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int index = BLOCK_WIDTH * bx + tx;

    // Load input to shared memory, zero if out of bounds
    temp[tx] = (index < n) ? d_in[index] : 0;
    __syncthreads();

    // Perform the actual scan using a more optimal parallel approach
    for(int offset = 1; offset < BLOCK_WIDTH; offset <<= 1) {
        unsigned int val = (tx >= offset) ? temp[tx - offset] : 0;
        __syncthreads();
        temp[tx] += val;
        __syncthreads();
    }

    // Write results to d_out and set d_out[0] to 0
    if(index + 1 < n) {
        d_out[index + 1] = temp[tx];
    }
    if(index == 0) {
        d_out[0] = 0;
    }

    // Store the total sum of each block
    if(tx == BLOCK_WIDTH - 1) {
        d_total[bx] = temp[tx];
    }
}