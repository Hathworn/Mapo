#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void partialScan(unsigned int *d_in, unsigned int *d_out, unsigned int *d_total, size_t n)
{
    __shared__ unsigned int temp[BLOCK_WIDTH];
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int index = BLOCK_WIDTH * bx + tx;

    // Load input to shared memory with boundary check
    temp[tx] = (index < n) ? d_in[index] : 0;
    __syncthreads();

    // Use binary tree method for scan
    for(int offset = 1; offset < BLOCK_WIDTH; offset <<= 1) {
        unsigned int temp_val = (tx >= offset) ? temp[tx - offset] : 0;
        __syncthreads();
        temp[tx] += temp_val;
        __syncthreads();
    }

    // Copy results for exclusive scan with boundary check
    if(index + 1 < n) {
        d_out[index + 1] = temp[tx];
    }
    if(index == 0) {
        d_out[0] = 0;
    }

    // Store the block sum using the last element
    if(tx == BLOCK_WIDTH - 1) {
        d_total[bx] = temp[BLOCK_WIDTH - 1];
    }
}