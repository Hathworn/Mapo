#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mapScan(unsigned int *d_array, unsigned int *d_total, size_t n) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int index = BLOCK_WIDTH * bx + tx;

    if(index < n) {
        // Use shared memory to reduce global memory access latency
        __shared__ unsigned int total;
        if(tx == 0) {
            total = d_total[bx];
        }
        __syncthreads();
        
        d_array[index] += total;
    }
}