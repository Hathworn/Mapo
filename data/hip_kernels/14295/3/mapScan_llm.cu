#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mapScan(unsigned int *d_array, unsigned int *d_total, size_t n) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int index = BLOCK_WIDTH * bx + tx;

    // Ensure index access is within array bounds. Pre-calculate total for improved performance.
    if (index < n) {
        unsigned int total = d_total[bx];
        d_array[index] += total;
    }
}