#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize using shared memory to reduce global memory access latency
__global__ void mapPredicate(unsigned int *d_zeros, unsigned int *d_ones, unsigned int *d_in, unsigned int bit, size_t n)
{
    extern __shared__ unsigned int sharedData[];  // allocate shared memory
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int index = BLOCK_WIDTH * bx + tx;

    if(index < n) {
        sharedData[tx] = d_in[index];  // load data into shared memory
        __syncthreads();  // ensure all data is loaded before processing
        
        unsigned int isOne = (sharedData[tx] >> bit) & 1;  // compute using shared memory
        d_ones[index] = isOne;
        d_zeros[index] = 1 - isOne;
    }
}