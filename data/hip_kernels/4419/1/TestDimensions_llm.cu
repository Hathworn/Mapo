#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TestDimensions()
{
    // Cache grid and block dimensions
    int gridX = gridDim.x;
    int gridY = gridDim.y;
    int gridZ = gridDim.z;
    int blockX = blockDim.x;
    int blockY = blockDim.y;
    int blockZ = blockDim.z;

    // Compute constants
    int const num_threads_per_block = blockX * blockY * blockZ;
    int const num_blocks = gridX * gridY * gridZ;
    int const threads_stride = num_threads_per_block * num_blocks;

    // Optimize thread_id calculation using cached dimensions
    int const thread_id = blockIdx.x * num_threads_per_block +
                          blockIdx.y * gridX * num_threads_per_block +
                          blockIdx.z * gridX * gridY * num_threads_per_block +
                          threadIdx.x +
                          threadIdx.y * blockX +
                          threadIdx.z * blockX * blockY;

    if (thread_id == 0)
    {
        printf("gridDim   = x: %6d / y: %6d / z: %6d\r\n", gridX, gridY, gridZ);
        printf("blockDim  = x: %6d / y: %6d / z: %6d\r\n", blockX, blockY, blockZ);
        printf("num_threads_per_block: %6d\r\n", num_threads_per_block);
        printf("num_blocks           : %6d\r\n", num_blocks);
        printf("threads_stride       : %6d\r\n", threads_stride);
    }

    printf("tidx | %6d | %6d | %6d | bidx | %6d | %6d | %6d | "
           "gdim | %6d | %6d | %6d | bdim | %6d | %6d | %6d | "
           "thread_id | %6d |\r\n",
           threadIdx.x, threadIdx.y, threadIdx.z,
           blockIdx.x, blockIdx.y, blockIdx.z,
           gridX, gridY, gridZ,
           blockX, blockY, blockZ, thread_id);

    return;
}