#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Global variables
int N = 93, dimx = 1920, dimy = 2560, tam_imag = 1920 * 2560;

// Kernel function
__global__ void kernel(float *max, float *var, int *top, int k) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Unroll comparison and assignment for better performance
    if (idx < tam_imag && var[idx] > max[idx]) {
        top[idx] = k;
        max[idx] = var[idx];
    }
}

// Additional functions can go here
float *leerMatrizVarianza(int d);