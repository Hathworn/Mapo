#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel to divide accuracy by N
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use a single thread to perform the division
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}