#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_rand_states_kernel(hiprandState *state, int seed)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
/* Each thread gets same seed, a different sequence
number, no offset */
hiprand_init(seed, id, 0, &state[id]);
}