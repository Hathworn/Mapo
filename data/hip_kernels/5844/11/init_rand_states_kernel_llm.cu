#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_rand_states_kernel(hiprandState *state, int seed)
{
    // Compute the global thread ID
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Check if the thread ID is within bounds for safety
    if (id >= gridDim.x * blockDim.x) return;

    // Initialize random state with unique sequence for each thread
    hiprand_init(seed, id, 0, &state[id]);
}