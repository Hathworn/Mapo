#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
typedef enum color {BLUE, GREEN, RED} Color;

__global__ void convert_to_rgb(float *hsv, unsigned char *dest, int width, int height, int step, int channels) {
    // Calculate global index for processing
    int ren = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within bounds
    if (ren < height && col < width) {
        float r, g, b;
        float h = hsv[(ren * step) + (col * channels) + RED];
        float s = hsv[(ren * step) + (col * channels) + GREEN];
        float v = hsv[(ren * step) + (col * channels) + BLUE];

        // use efficient modulo and division
        float f = h / 60.0f;
        float hi = __float2int_rd(f) % 6;
        f = f - hi;
        float p = v * (1 - s);
        float q = v * (1 - s * f);
        float t = v * (1 - s * (1 - f));

        // Simplified assignment using switch-case
        switch ((int)hi) {
            case 0: r = v; g = t; b = p; break;
            case 1: r = q; g = v; b = p; break;
            case 2: r = p; g = v; b = t; break;
            case 3: r = p; g = q; b = v; break;
            case 4: r = t; g = p; b = v; break;
            case 5: r = v; g = p; b = q; break;
        }

        // Efficient conversion and assignment
        dest[(ren * step) + (col * channels) + RED] = (unsigned char) __float2uint_rn(255.0f * r);
        dest[(ren * step) + (col * channels) + GREEN] = (unsigned char) __float2uint_rn(255.0f * g);
        dest[(ren * step) + (col * channels) + BLUE] = (unsigned char) __float2uint_rn(255.0f * b);
    }
}