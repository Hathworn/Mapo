#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertBGR2RGBfloatKernel(uchar3 *src, float3 *dst, int width, int height)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Use a single condition check to cover both x and y bounds
    if (x < width && y < height) {
        int idx = y * width + x;  // Calculate index once
        uchar3 color = src[idx];
        dst[idx] = make_float3(color.z, color.y, color.x);
    }
}