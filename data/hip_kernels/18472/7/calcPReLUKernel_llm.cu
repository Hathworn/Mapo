#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcPReLUKernel(const float *input, float *output, const float *weights, int width, int height, int channels)
{
    // Compute global thread index
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Calculate the linear index for input and output arrays
    int index = y * width + x;

    // Ensure the thread is within bounds
    if (x < width && y < height) {
        // Apply PReLU activation
        float inpVal = input[index];
        output[index] = inpVal > 0 ? inpVal : inpVal * weights[y % channels];
    }
}