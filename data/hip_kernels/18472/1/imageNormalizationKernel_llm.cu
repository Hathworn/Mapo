#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void imageNormalizationKernel(float3 *ptr, int width, int height)
{
    // Calculate global thread index
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Exit if thread is out of bounds
    if (x >= width || y >= height) {
        return;
    }

    // Pre-compute pixel index
    int pixelIdx = y * width + x;

    // Load color at pixelIdx
    float3 color = ptr[pixelIdx];

    // Normalize color components
    float normalizationFactor = 0.0078125;
    color.x = (color.x - 127.5) * normalizationFactor;
    color.y = (color.y - 127.5) * normalizationFactor;
    color.z = (color.z - 127.5) * normalizationFactor;

    // Store normalized color back to memory
    ptr[pixelIdx] = color;
}