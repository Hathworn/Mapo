#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void imageSplitKernel(float3 *ptr, float *dst, int width, int height)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Check if within image bounds
    if (x < width && y < height) {
        int idx = y * width + x; // Calculate base index once
        float3 color = ptr[idx]; // Optimize memory access
        
        dst[idx] = color.x; // Store R channel
        dst[idx + width * height] = color.y; // Store G channel
        dst[idx + 2 * width * height] = color.z; // Store B channel
    }
}