#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void imagePaddingKernel(float3 *ptr, float3 *dst, int width, int height, int top, int bottom, int left, int right)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Check if the thread is within bounds of the padded image
    if (x >= left && x < (width - right) && y >= top && y < (height - bottom)) {
        
        // Calculate the source position considering padding
        int srcX = x - left;
        int srcY = y - top;
        int paddedWidth = width - right - left;

        // Load the color from source and store in destination
        float3 color = ptr[srcY * paddedWidth + srcX];
        dst[y * width + x] = color;
    }
}