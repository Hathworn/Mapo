#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if idx is within the bounds to improve performance
    if (idx < numEls) {
        dest[idx] = a[idx] + b[idx];
    }
}