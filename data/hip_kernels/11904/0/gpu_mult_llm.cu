#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define B 1 // blocks in the grid
#define T 10 // threads in a block

#ifdef BAMBOO_PROFILING
#else
#endif

__global__ void gpu_mult(int *a, int *b, int *c, int N) {

    // Use shared memory for better data reuse and to minimize global memory accesses
    __shared__ int shared_a[T][T];
    __shared__ int shared_b[T][T];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;

    for (int tile = 0; tile < (N + T - 1) / T; ++tile) {
        // Load elements from global to shared memory
        if (row < N && tile * T + threadIdx.x < N) {
            shared_a[threadIdx.y][threadIdx.x] = a[row * N + tile * T + threadIdx.x];
        } else {
            shared_a[threadIdx.y][threadIdx.x] = 0;
        }

        if (tile * T + threadIdx.y < N && col < N) {
            shared_b[threadIdx.y][threadIdx.x] = b[(tile * T + threadIdx.y) * N + col];
        } else {
            shared_b[threadIdx.y][threadIdx.x] = 0;
        }

        // Synchronize to ensure all data is loaded to shared memory
        __syncthreads();

        // Compute partial sum for this tile
        for (int i = 0; i < T; ++i) {
            sum += shared_a[threadIdx.y][i] * shared_b[i][threadIdx.x];
        }

        // Ensure all threads have completed computation for this tile
        __syncthreads();
    }

    // Write the result to global memory
    if (row < N && col < N) {
        c[row * N + col] = sum;
    }
}