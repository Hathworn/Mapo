#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shiftRightPixels(int16_t *bayImg, size_t width, size_t height, int bppMult)
{
    // Calculate the linear index directly to avoid recomputation
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height)
    {
        // Use linear indexing for direct memory access
        bayImg[y * width + x] >>= bppMult;
    }
}