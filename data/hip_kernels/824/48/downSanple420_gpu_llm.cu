#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

__global__ void downSanple420_gpu(hipTextureObject_t ch1, hipTextureObject_t ch2, int16_t *downCh1, int16_t *downCh2, size_t width, size_t height)
{
    int2 threadCoord = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    if (threadCoord.x < width && (threadCoord.y << 1) < height)
    {
        int2 pixelCoord;
        hipTextureObject_t *ch;
        int16_t *downCh;

        // Optimize the condition check by using compact form and avoid redundancy
        bool isCh1 = threadCoord.x < (width >> 1);
        pixelCoord = make_int2((threadCoord.x - (isCh1 ? 0 : (width >> 1))) << 1, threadCoord.y << 1);
        ch = isCh1 ? &ch1 : &ch2;
        downCh = isCh1 ? downCh1 : downCh2;

        int16_t bias = (pixelCoord.x & 1) + 1;

        // Optimize memory access by reducing operational calls inside tex2D
        int16_t pixel =
            (tex2D<int16_t>(*ch, pixelCoord.x, pixelCoord.y) +
             tex2D<int16_t>(*ch, pixelCoord.x + 1, pixelCoord.y) +
             tex2D<int16_t>(*ch, pixelCoord.x, pixelCoord.y + 1) +
             tex2D<int16_t>(*ch, pixelCoord.x + 1, pixelCoord.y + 1) +
             bias) >> 2;
        
        downCh[(pixelCoord.y >> 1) * width + (pixelCoord.x >> 1)] = pixel;
    }
}