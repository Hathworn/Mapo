#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void downSanple420_gpu(hipTextureObject_t ch1, hipTextureObject_t ch2, uint8_t *downCh1, uint8_t *downCh2, size_t width, size_t height)
{
    int2 threadCoord = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    if (threadCoord.x < width && (threadCoord.y << 1) < height)
    {
        int2 pixelCoord = make_int2((threadCoord.x % (width >> 1)) << 1, threadCoord.y << 1); // Consolidate pixelCoord calculation
        hipTextureObject_t *ch = (threadCoord.x < (width >> 1)) ? &ch1 : &ch2; // Use conditional operator
        uint8_t *downCh = (threadCoord.x < (width >> 1)) ? downCh1 : downCh2;  // Use conditional operator

        int16_t bias = (pixelCoord.x & 1) + 1;
        uint16_t pixel = (tex2D<uint16_t>(*ch, pixelCoord.x, pixelCoord.y)
                        + tex2D<uint16_t>(*ch, pixelCoord.x + 1, pixelCoord.y)
                        + tex2D<uint16_t>(*ch, pixelCoord.x, pixelCoord.y + 1)
                        + tex2D<uint16_t>(*ch, pixelCoord.x + 1, pixelCoord.y + 1) + bias) >> 2;

        downCh[((pixelCoord.y >> 1) * width + pixelCoord.x) >> 1] = (uint8_t)pixel;
    }
}