#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void downSanple422_gpu(hipTextureObject_t ch1, hipTextureObject_t ch2, uint8_t *downCh1, uint8_t *downCh2, size_t width, size_t height)
{
    // Calculate global thread position
    int2 threadCoord = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    
    // Only process if within bounds
    if (threadCoord.x < width && threadCoord.y < height)
    {
        int2 pixelCoord;
        hipTextureObject_t currentChannel;
        uint8_t *downCh;

        // Determine which half and setup pointers accordingly
        if (threadCoord.x < (width >> 1))
        {
            pixelCoord = make_int2(threadCoord.x << 1, threadCoord.y);
            currentChannel = ch1;
            downCh = downCh1;
        }
        else
        {
            pixelCoord = make_int2((threadCoord.x - (width >> 1)) << 1, threadCoord.y);
            currentChannel = ch2;
            downCh = downCh2;
        }

        // Calculate pixel value
        int16_t bias = pixelCoord.x & 1;
        uint16_t leftPixel = tex2D<uint16_t>(currentChannel, pixelCoord.x, pixelCoord.y);
        uint16_t rightPixel = tex2D<uint16_t>(currentChannel, pixelCoord.x + 1, pixelCoord.y);
        uint16_t pixel = (leftPixel + rightPixel + bias) >> 1;
        
        // Store result in downsampled channel
        downCh[(pixelCoord.y * width + pixelCoord.x) >> 1] = (uint8_t)pixel;
    }
}