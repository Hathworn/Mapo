#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void downSanple422_gpu(hipTextureObject_t ch1, hipTextureObject_t ch2, int16_t *downCh1, int16_t *downCh2, size_t width, size_t height)
{
    int2 threadCoord = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    if (threadCoord.x < width && threadCoord.y < height)
    {
        int2 pixelCoord;
        hipTextureObject_t *ch;
        int16_t *downCh;

        // Optimize warp divergence by using even-odd division directly
        if (threadCoord.x & (width >> 1)) 
        {
            pixelCoord = make_int2((threadCoord.x - (width >> 1)) << 1, threadCoord.y);
            ch = &ch2;
            downCh = downCh2;
        }
        else
        {
            pixelCoord = make_int2(threadCoord.x << 1, threadCoord.y);
            ch = &ch1;
            downCh = downCh1;
        }
        
        int16_t bias = pixelCoord.x & 1;
        int16_t pixel = (tex2D<int16_t>(*ch, pixelCoord.x, pixelCoord.y) + tex2D<int16_t>(*ch, pixelCoord.x + 1, pixelCoord.y) + bias) >> 1;
        downCh[pixelCoord.y * width + (pixelCoord.x >> 1)] = pixel;
    }
}