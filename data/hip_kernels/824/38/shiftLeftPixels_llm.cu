#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shiftLeftPixels(int16_t *bayImg, size_t width, size_t height, int bppMult)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height)
    {
        // Calculate the index once for efficiency
        int idx = y * width + x;
        // Perform bit shift with pre-computed index
        bayImg[idx] <<= bppMult;
    }
}