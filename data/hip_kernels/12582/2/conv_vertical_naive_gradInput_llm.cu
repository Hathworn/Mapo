#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv_vertical_naive_gradInput(const int n, float *dx, const float *dy, const float *w, const int oH, const int oW, const int kL)
{
    // Each thread calculates a unique index within the bounds of data size n
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int gridSize = blockDim.x * gridDim.x;

    // Loop to accommodate large data sizes
    while (i < n) { 
        int iH = oH + kL - 1;
        int iC = i / (iH * oW);
        int row = (i % (iH * oW)) / oW;
        int dy_offset = iC * oH * oW + i % (iH * oW);
        int w_offset = iC * kL;

        int k_begin = max(0, row - oH + 1);
        int k_end = min(kL, row + 1);

        float result = 0.0f; // Accumulate results in a temporary variable
        for (int k = k_begin; k < k_end; k++) {
            result += w[w_offset + k] * dy[dy_offset - k * oW];
        }
        dx[i] = result; // Assign accumulated result to output

        i += gridSize; // Move to next index
    }
}