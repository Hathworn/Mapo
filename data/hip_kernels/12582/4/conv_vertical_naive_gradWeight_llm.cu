#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv_vertical_naive_gradWeight(const int n, float *y, const float *x, const int kL, const int iC)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = blockDim.x * gridDim.x;
    
    // Use strided loop for better GPU utilization
    for (; i < n; i += gridStride) {
        int idx = (i / kL) * kL * iC + i; // Avoid recalculating index every iteration
        y[i] = x[idx];
    }
}