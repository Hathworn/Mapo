#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv_horizontal_naive_output(const int n, float *y, const float *x, const float *w, const int iH, const int iW, const int kL)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for tile-based optimization
    __shared__ float shared_w[256];  // Assuming kL <= 256

    // Precompute values that are constant in this iteration
    int oW = iW - kL + 1;
    
    // Iterate over data by stride
    while (i < n) {
        int x_offset = (i / oW) * iW + (i % oW);
        int w_offset = (i / (oW * iH)) * kL;

        // Load weights into shared memory
        if (threadIdx.x < kL) {
            shared_w[threadIdx.x] = w[w_offset + threadIdx.x];
        }
        
        // Ensure all threads have loaded weights
        __syncthreads();
        
        // Convolution calculation
        float result = 0.0f;
        for (int k = 0; k < kL; k++) {
            result += shared_w[k] * x[x_offset + k];
        }
        y[i] = result;
        
        // Move to next data point
        i += blockDim.x * gridDim.x;
    }
}