#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv_vertical_naive_output(const int n, float *y, const float *x, const float *w, const int iH, const int iW, const int kL)
{
    // Precompute constants outside of loop
    int oH = iH - kL + 1;
    int stride = blockDim.x * gridDim.x;
    
    // Use shared memory to reduce global memory accesses
    __shared__ float shared_w[1024]; // Size might need adjustment based on kL

    // Each thread calculates its work index
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += stride) {

        int x_offset = (i / (oH * iW)) * iH * iW + i % (oH * iW);
        int w_offset = (i / (oH * iW)) * kL;

        // Load weights into shared memory
        if (threadIdx.x < kL) {
            shared_w[threadIdx.x] = w[w_offset + threadIdx.x];
        }
        __syncthreads(); // Ensure all weights are loaded before proceeding

        float sum = 0.0f; // Temporary variable to accumulate results
        for (int k = 0; k < kL; k++) {
            sum += shared_w[k] * x[x_offset + k * iW];
        }
        y[i] = sum; // Store result back to global memory
    }
}