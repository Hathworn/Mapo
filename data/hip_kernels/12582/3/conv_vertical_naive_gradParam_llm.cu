#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv_vertical_naive_gradParam(const int n, float *dw, const float *x, const float *dy, const int kL, const int oH, const int oW)
{
    // Calculate the global index based on block and thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Iterate over grid-stride loop to cover all elements
    for (; i < n; i += blockDim.x * gridDim.x) {
        // Calculate offsets for x and dy arrays
        int dy_offset = (i / kL) * oH * oW;
        int x_offset = (i / kL) * oH * oW + (i % kL) * oW;

        // Use inner-loop unrolling to reduce loop overhead
        float sum = 0.0f;
        for (int k = 0; k < oH * oW; k++) {
            sum += dy[dy_offset + k] * x[x_offset + k];
        }
        dw[i] = sum;
    }
}