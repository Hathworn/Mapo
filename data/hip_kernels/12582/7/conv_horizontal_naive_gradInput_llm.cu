#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv_horizontal_naive_gradInput(const int n, float *dx, const float *dy, const float *w, const int oH, const int oW, const int kL)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = blockDim.x * gridDim.x;  // Total number of threads

    // Use a loop with stride equal to the total number of threads
    while (i < n) {
        int iW = oW + kL - 1;
        int col = i % iW;
        int dy_offset = (i / iW) * oW + i % iW;
        int w_offset = (i / (iW * oH)) * kL;

        int k_begin = max(0, col - oW + 1);
        int k_end = min(kL, col + 1);

        dx[i] = 0.0f;
        for (int k = k_begin; k < k_end; k++) {
            dx[i] += w[w_offset + k] * dy[dy_offset - k];
        }

        i += totalThreads;  // Increment by total number of threads
    }
}