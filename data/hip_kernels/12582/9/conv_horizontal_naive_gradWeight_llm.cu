#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv_horizontal_naive_gradWeight(const int n, float *y, const float *x, const int kL, const int iC)
{
    // Improved to reduce division operation within the loop
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += totalThreads) {
        int baseIndex = (i / kL) * kL * iC;  // Calculate base index outside the loop
        y[i] = x[baseIndex + i];
    }
}