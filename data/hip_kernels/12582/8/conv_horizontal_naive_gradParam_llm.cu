#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv_horizontal_naive_gradParam(const int n, float *dw, const float *x, const float *dy, const int kL, const int oH, const int oW)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) { // Avoid out-of-bounds access
        int iW = oW + kL - 1;
        int dy_offset = (i / kL) * oH * oW;
        int x_offset = (i / kL) * oH * oW + i % kL;
        float temp_dw = 0.0f; // Use a temporary variable to reduce memory access

        for (int j = 0; j < oH; j++) {
            for (int k = 0; k < oW; k++) {
                temp_dw += dy[dy_offset + j * oW + k] * x[x_offset + j * iW + k];
            }
        }
        dw[i] = temp_dw; // Write back once per thread
    }
}