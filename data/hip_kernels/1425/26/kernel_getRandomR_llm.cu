#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_getRandomR(double* dev_mat, double* dev_ramR, int* dev_nc)
{
    // Shared memory optimization
    extern __shared__ double matS[];

    // Calculate thread id
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Check if insertion should occur for this image
    if (threadIdx.x < dev_nc[blockIdx.x]) 
    {
        double *mat = matS + threadIdx.x * 18;
        double *res = matS + threadIdx.x * 18 + 9;  // Use shared memory for res
        
        // Initialize mat elements
        mat[0] = 0; mat[4] = 0; mat[8] = 0;
        mat[5] = dev_ramR[tid * 4 + 1];
        mat[6] = dev_ramR[tid * 4 + 2];
        mat[1] = dev_ramR[tid * 4 + 3];
        mat[7] = -mat[5];
        mat[2] = -mat[6];
        mat[3] = -mat[1];

        // Clear res elements using shared memory
        for(int i = 0; i < 9; i++)
            res[i] = 0;

        // Matrix multiplication optimization
        for (int i = 0; i < 3; i++)
        {
            for (int j = 0; j < 3; j++)
            {
                double sum = 0;  // Use local variable for sum
                for (int k = 0; k < 3; k++)
                {
                    sum += mat[i + k * 3] * mat[k + j * 3];
                }
                res[i + j * 3] = sum;
            }
        }

        double scale = 2.0 * dev_ramR[tid * 4];
        
        // Perform scaling and sum operations
        for (int n = 0; n < 9; n++)
        {
            mat[n] *= scale;
            mat[n] += res[n] * 2.0;
        }

        // Add identity matrix component
        mat[0] += 1.0;
        mat[4] += 1.0;
        mat[8] += 1.0;

        // Write result to global memory
        for (int n = 0; n < 9; n++)
        {
            dev_mat[tid * 9 + n] = mat[n];
        }
    }
}