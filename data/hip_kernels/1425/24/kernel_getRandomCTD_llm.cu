#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_getRandomCTD(double* dev_nt, double* dev_tran, double* dev_nd, double* dev_ramD, double* dev_nr, double* dev_ramR, unsigned int out, int rSize, int tSize, int dSize )
{
    // Calculate global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Initialize random state
    hiprandState s;
    hiprand_init(out, tid, 0, &s);

    // Optimize random coordinate calculation for tran
    float myrand = hiprand_uniform(&s);
    int t = ((int)truncf(myrand * tSize) + blockIdx.x * tSize) * 2;
    for (int n = 0; n < 2; n++)
    {
        dev_tran[tid * 2 + n] = dev_nt[t + n];
    }

    // Optimize random coordinate calculation for ramR
    myrand = hiprand_uniform(&s);
    int r = ((int)truncf(myrand * rSize) + blockIdx.x * rSize) * 4;
    for (int n = 0; n < 4; n++)
    {
        dev_ramR[tid * 4 + n] = dev_nr[r + n];
    }

    // Optimize random coordinate calculation for ramD
    myrand = hiprand_uniform(&s);
    dev_ramD[tid] = dev_nd[blockIdx.x * dSize + (int)truncf(myrand * dSize)];
}