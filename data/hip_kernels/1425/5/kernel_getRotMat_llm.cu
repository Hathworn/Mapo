#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_getRotMat(double* devRotm, double* devnR, int nR)
{
    extern __shared__ double matS[];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= nR)
        return;

    double *mat, *res;
    mat = matS + threadIdx.x * 18;
    res = mat  + 9;

    // Init shared memory matrices
    mat[0] = 0; mat[4] = 0; mat[8] = 0;
    mat[5] = devnR[tid * 4 + 1];
    mat[6] = devnR[tid * 4 + 2];
    mat[1] = devnR[tid * 4 + 3];
    mat[7] = -mat[5];
    mat[2] = -mat[6];
    mat[3] = -mat[1];

    // Initialize res matrix to 0
    #pragma unroll
    for(int i = 0; i < 9; i++)
        res[i] = 0;

    // Matrix multiplication with loop unrolling
    #pragma unroll
    for (int i = 0; i < 3; i++)
        #pragma unroll
        for (int j = 0; j < 3; j++)
            #pragma unroll
            for (int k = 0; k < 3; k++)
                res[i + j * 3] += mat[i + k * 3] * mat[k + j * 3];

    double scale = 2 * devnR[tid * 4];
    #pragma unroll
    for (int n = 0; n < 9; n++)
    {
        mat[n] = mat[n] * scale + res[n] * 2;
    }

    // Adjust diagonal elements
    mat[0] += 1;
    mat[4] += 1;
    mat[8] += 1;

    // Store result in global memory
    #pragma unroll
    for (int n = 0; n < 9; n++)
    {
        devRotm[tid * 9 + n] = mat[n];
    }
}