#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_getRandomCTD(double* dev_nt, double* dev_tran, double* dev_nr, double* dev_ramR, unsigned int out, int rSize, int tSize )
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandState s;
    hiprand_init(out, tid, 0, &s);

    // Get random index for dev_tran
    float myrand_t = hiprand_uniform(&s) * tSize;
    int t = ((int)truncf(myrand_t) + blockIdx.x * tSize) * 2;
    for (int n = 0; n < 2; n++)
    {
        dev_tran[tid * 2 + n] = dev_nt[t + n];
    }

    // Get random index for dev_ramR
    float myrand_r = hiprand_uniform(&s) * rSize;
    int r = ((int)truncf(myrand_r) + blockIdx.x * rSize) * 4;
    for (int n = 0; n < 4; n++)
    {
        dev_ramR[tid * 4 + n] = dev_nr[r + n];
    }
}