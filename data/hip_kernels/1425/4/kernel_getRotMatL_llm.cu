#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_getRotMatL(double* devRotm, double* devnR, int nR)
{
    extern __shared__ double matS[];

    double *mat = matS + threadIdx.x * 18;
    double *res = mat + 9;

    mat[0] = mat[4] = mat[8] = 0;
    mat[5] = devnR[threadIdx.x * 4 + 1];
    mat[6] = devnR[threadIdx.x * 4 + 2];
    mat[1] = devnR[threadIdx.x * 4 + 3];
    mat[7] = -mat[5];
    mat[2] = -mat[6];
    mat[3] = -mat[1];

    // Initialize res array with zero
    #pragma unroll
    for(int i = 0; i < 9; i++) res[i] = 0;

    // Matrix multiplication using manual unrolling
    #pragma unroll
    for (int i = 0; i < 3; i++)
    {
        #pragma unroll
        for (int j = 0; j < 3; j++)
        {
            #pragma unroll
            for (int k = 0; k < 3; k++)
            {
                res[i * 3 + j] += mat[i * 3 + k] * mat[k * 3 + j];
            }
        }
    }

    double scale = 2 * devnR[threadIdx.x * 4];
    #pragma unroll
    for (int n = 0; n < 9; n++)
    {
        mat[n] = mat[n] * scale + res[n] * 2;
    }

    mat[0] += 1;
    mat[4] += 1;
    mat[8] += 1;

    #pragma unroll
    for (int n = 0; n < 9; n++)
    {
        devRotm[threadIdx.x * 9 + n] = mat[n];
    }
}