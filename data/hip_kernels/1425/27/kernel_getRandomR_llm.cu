#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_getRandomR(double* dev_mat, double* dev_ramR)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    extern __shared__ double matS[];

    double *mat, *res;
    mat = matS + threadIdx.x * 18;
    res = mat + 9;

    // Initialize mat array using a loop for cleanliness
    for(int i = 0; i < 9; i++) {
        mat[i] = 0;
    }
    mat[5] = dev_ramR[tid * 4 + 1];
    mat[6] = dev_ramR[tid * 4 + 2];
    mat[1] = dev_ramR[tid * 4 + 3];
    mat[7] = -mat[5];
    mat[2] = -mat[6];
    mat[3] = -mat[1];

    // Initialize res array using a loop for cleanliness
    for(int i = 0; i < 9; i++) {
        res[i] = 0;
    }

    // Optimized matrix multiplication using single loop
    #pragma unroll
    for (int i = 0; i < 9; i++)
        res[i] = mat[i%3] * mat[i/3*3] + mat[i%3+3] * mat[i/3*3+1] + mat[i%3+6] * mat[i/3*3+2];

    double scale = 2 * dev_ramR[tid * 4];
    #pragma unroll
    for (int n = 0; n < 9; n++) {
        mat[n] = mat[n] * scale + res[n] * 2;
    }

    mat[0] += 1;
    mat[4] += 1;
    mat[8] += 1;

    #pragma unroll
    for (int n = 0; n < 9; n++) {
        dev_mat[tid * 9 + n] = mat[n];
    }
}