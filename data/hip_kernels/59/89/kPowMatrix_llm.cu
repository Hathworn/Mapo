#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kPowMatrix(float* mat, float* pow, float* target, unsigned int len) {
    // Calculate global index based on block and thread indices
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop to process elements beyond the first
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        target[i] = powf(mat[i], pow[i]);
    }
}