#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function for scalar lower bound
__global__ void kLowerBoundScalar(float* mat, float val, float* target, unsigned int len) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    // Use strided loop for better memory coalescing
    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = fmaxf(mat[i], val); // Use fmaxf for potential performance gain
    }
}