#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByRowVectorScale(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height, float scale_targets) {
    const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;  // Calculate row index
    const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate column index

    if (row < height && col < width) {  // Ensure we are within matrix bounds
        const unsigned int idx = row * width + col;  // Calculate linear index for matrix
        tgtMat[idx] = scale_targets * tgtMat[idx] + mat[idx] * vec[row];  // Optimized computation
    }
}