#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kReciprocal(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    // Loop unrolling for better performance
    for (unsigned int i = idx; i + 4 <= len; i += 4 * stride) {
        target[i] = 1.0f / mat[i];
        if (i + stride < len) target[i + stride] = 1.0f / mat[i + stride];
        if (i + 2 * stride < len) target[i + 2 * stride] = 1.0f / mat[i + 2 * stride];
        if (i + 3 * stride < len) target[i + 3 * stride] = 1.0f / mat[i + 3 * stride];
    }
}