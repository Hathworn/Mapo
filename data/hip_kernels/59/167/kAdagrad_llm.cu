#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdagrad(float *history, float *grad, float delta, int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Use shared memory for faster square root computation
    extern __shared__ float shared_grad[];

    for (unsigned int i = idx; i < len; i += numThreads) {
        float curr_norm = history[i] - delta;
        shared_grad[threadIdx.x] = grad[i];
        __syncthreads();  // Ensure all threads have loaded data

        history[i] = delta + sqrtf(curr_norm * curr_norm + shared_grad[threadIdx.x] * shared_grad[threadIdx.x]);
        __syncthreads();  // Ensure all threads have finished writing to history
    }
}