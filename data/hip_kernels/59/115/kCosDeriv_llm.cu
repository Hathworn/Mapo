#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCosDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    // Optimize using shared memory for temporary storage
    __shared__ float sharedA[256]; // Adjust size based on blockDim.x

    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        // Load into shared memory
        sharedA[threadIdx.x] = -a[i];

        // Ensure all threads have written to shared memory
        __syncthreads();

        dest[i] = sharedA[threadIdx.x] * __sinf(b[i]);

        // Ensure all threads have read from shared memory before next iteration
        __syncthreads();
    }
}