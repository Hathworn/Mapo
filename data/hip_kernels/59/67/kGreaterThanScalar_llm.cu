#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kGreaterThanScalar(float* mat, float val, float* target, unsigned int len) {
    // Calculate the global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop to cover the entire array
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        target[i] = mat[i] > val; // Direct computation without branching
    }
}