#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kUpperBound(float* mat1, float* mat2, float* target, unsigned int len) {
    // Calculate global thread ID
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index is within bounds before accessing data
    if (idx < len) {
        target[idx] = mat1[idx] > mat2[idx] ? mat2[idx] : mat1[idx];
    }
}