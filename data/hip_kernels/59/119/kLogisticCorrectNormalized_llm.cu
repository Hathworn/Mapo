#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogisticCorrectNormalized(float* mat, float* targets, float* out, unsigned int height, unsigned int width) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < height) {
        float correct = 0;
        float total = 0;

        // Use register variables for p, t to reduce memory access latency.
        for (int i = idx; i < width * height; i += height) {
            float p = mat[i];
            float t = targets[i];
            // Remove __syncthreads() as it's unnecessary here; no shared memory used.
            correct += (t < 0) ? 0 : (((t >= 0.5 && p >= 0.5) || (t < 0.5 && p < 0.5)) ? 1 : 0);
            total += (t < 0) ? 0 : 1;
        }

        out[idx] = (total > 0) ? (correct / total) : 0;
    }
}