#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kExtractPatches2(float* images, float* patches, float* width_offset, float* height_offset, float* flip, int num_images, int img_width, int img_height, int patch_width, int patch_height, int num_colors) {
    // Calculate image, color, and destination indices more efficiently
    int image_id = blockIdx.z % num_images;
    int color = blockIdx.z / num_images;
    int dest_col = blockIdx.x * blockDim.x + threadIdx.x;
    int dest_row = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure threads operate only within valid patch dimensions
    if (dest_col < patch_width && dest_row < patch_height) {
        // Calculate source position with offset adjustments
        int source_row = static_cast<int>(height_offset[image_id]) + dest_row;
        int source_col = static_cast<int>(width_offset[image_id]) + dest_col;
        // Apply flip adjustment
        if (flip[image_id] > 0.5f) {
            source_col = img_width - source_col - 1;
        }

        // Calculate linear indices for source and destination
        unsigned long dest_index = image_id + num_images * (dest_col + patch_width * (dest_row + patch_height * color));
        unsigned long source_index = source_col + img_width * (source_row + img_height * (color + num_colors * image_id));

        // Copy image data into patches
        patches[dest_index] = images[source_index];
    }
}