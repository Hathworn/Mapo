#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByColVector(float* __restrict__ mat, float* __restrict__ vec, float* __restrict__ tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = gridDim.x * blockDim.x;

    for (unsigned int i = idx; i < width * height; i += stride) {
        // Load vector element into a register to reduce global memory access
        float vecElement = vec[i % height];  
        tgtMat[i] = mat[i] * vecElement;
    }
}