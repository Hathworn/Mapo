#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kGenerateTranslationsBigVarOff(float* source, float* target, float* off_x_arr, float* off_y_arr, int source_w, int target_w, int num_channels) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; // Use global thread index
    const unsigned int numThreads = gridDim.x * blockDim.x; // Total number of threads

    int pad = (source_w - target_w) / 2;
    int target_tile_size = target_w * target_w;
    int source_tile_size = source_w * source_w;

    int off_x = off_x_arr[blockIdx.x];
    int off_y = off_y_arr[blockIdx.x];
    int target_off = blockIdx.x * target_tile_size;
    int source_off = blockIdx.x * source_tile_size + (pad + off_x) * source_w + (pad + off_y);

    for (unsigned int target_ind = idx; target_ind < target_tile_size; target_ind += numThreads) {
        int target_x = target_ind / target_w;
        int target_y = target_ind % target_w; // Optimize modulus operation

        int target_idx_base = target_off + target_x * target_w + target_y;
        int source_idx_base = source_off + target_x * source_w + target_y;

        for (unsigned int ch = 0; ch < num_channels; ch++) {
            target[num_channels * target_idx_base + ch] = source[num_channels * source_idx_base + ch];
        }
    }
}