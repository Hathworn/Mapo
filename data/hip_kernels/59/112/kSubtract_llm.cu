#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSubtract(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate the global index for each thread
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure threads work only within valid index range
    if (idx < numEls) {
        dest[idx] = a[idx] - b[idx];
    }
}