#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSqrt(float* mat, float* target, unsigned int len) {
    // Calculate the global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Optimize by unrolling the loop for better performance
    const unsigned int numThreads = blockDim.x * gridDim.x;
    for (unsigned int i = idx; i < len; i += numThreads) {
        target[i] = sqrt(mat[i]);
    }
}