#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultDiagonalScalar(float* mat, float val, float* tgtMat, unsigned int width) {
    // Compute unique thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure only threads within matrix size execute
    if (idx < width) {
        tgtMat[width * idx + idx] = mat[width * idx + idx] * val;
    }
}