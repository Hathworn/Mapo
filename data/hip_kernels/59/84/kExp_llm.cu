#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kExp(float* mat, float* target, unsigned int len) {
    // Use shared memory for better memory coalescing
    extern __shared__ float sharedMat[];
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    for (unsigned int i = idx; i < len; i += numThreads) {
        sharedMat[threadIdx.x] = mat[i];  // Load to shared memory
        __syncthreads();  // Synchronize threads to ensure all loads are complete
        target[i] = __expf(sharedMat[threadIdx.x]);  // Perform computation
        __syncthreads();  // Synchronize before next iteration
    }
}