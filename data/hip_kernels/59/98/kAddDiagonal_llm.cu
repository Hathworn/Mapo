#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddDiagonal(float* mat, float* vec, float* tgtMat, unsigned int width) {
    // Calculate global index based on block and thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within the range of the matrix width
    if (idx < width) {
        // Perform addition for the diagonal element specific to the index
        tgtMat[width * idx + idx] = mat[width * idx + idx] + vec[idx];
    }
}