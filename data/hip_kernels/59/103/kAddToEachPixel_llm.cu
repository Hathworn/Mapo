#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddToEachPixel(float* mat1, float* mat2, float* tgtMat, float mult, unsigned int width, unsigned int height, unsigned int num_pix) {
    // Calculate global index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Improved memory access pattern by reducing repeated calculations 
    unsigned int totalSize = width * height;
    unsigned int numThreads = blockDim.x * gridDim.x;
    unsigned int heightNumPix = height * num_pix;

    for (unsigned int i = idx; i < totalSize; i += numThreads) {
        unsigned int y = i % height;  // Column access optimization
        unsigned int x = i / heightNumPix;
        tgtMat[i] = mat1[i] + mult * mat2[y + height * x];
    }
}