#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kExpandAndAdd(float* source, float* mat, float* indices, float* target, int width, int height, float mult, int width2) {
    // Calculate a unique thread index for the kernel
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Total number of threads in the grid
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Loop over the data with stride equal to the number of threads
    for (unsigned int i = idx; i < width * height; i += numThreads) {
        // Efficiently compute position in the matrix using precomputed value
        const int index_height = indices[i / height];
        const int pos = height * static_cast<int>(index_height) + i % height;

        // Perform the calculation avoiding division by zero
        if (pos < height * width2) {
            target[i] = source[i] + mult * mat[pos];
        } else {
            target[i] = nanf("");  // Use NaN as error value
        }
    }
}