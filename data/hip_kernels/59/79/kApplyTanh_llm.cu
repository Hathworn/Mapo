#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplyTanh(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Unroll loop to improve performance
    for (unsigned int i = idx; i < len; i += numThreads) {
        float mat_i = mat[i];
        float exp2x = __expf(2.0f * mat_i); // Use 2.0f for improved precision
        target[i] = 1.0f - 2.0f / (exp2x + 1.0f); // Ensure consistent type usage with float
    }
}