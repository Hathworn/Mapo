#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kContract(float *expanded_data, float* targets, int num_images, int num_input_channels, int image_size_y, int image_size_x, int num_modules_y, int num_modules_x, int kernel_size_y, int kernel_size_x, int padding_y, int padding_x, int stride_y, int stride_x, int num_modules_batch, int module_id_offset) {
    int color = blockIdx.y;
    int dst_module_id = module_id_offset + blockIdx.x;
    int module_id_x = dst_module_id % num_modules_x;
    int module_id_y = dst_module_id / num_modules_x;
    int startX = module_id_x * stride_x + padding_x;
    int startY = module_id_y * stride_y + padding_y;
    long target_id, source_id;

    targets += num_images * image_size_x * image_size_y * color;
    expanded_data += num_images * blockIdx.x + num_modules_batch * kernel_size_y * kernel_size_x * color;

    // Unroll loops for better performance.
    for (int y = 0; y < kernel_size_y; y++) {
        int Y = startY + y;
        for (int x = 0; x < kernel_size_x; x++) {
            int X = startX + x;
            if (X >= 0 && X < image_size_x && Y >= 0 && Y < image_size_y) {
                source_id = num_images * (x + kernel_size_x * y);
                target_id = num_images * (X + image_size_x * Y);
                for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
                    atomicAdd(&targets[target_id + im], expanded_data[source_id + im]);
                }
            }
        }
    }
}