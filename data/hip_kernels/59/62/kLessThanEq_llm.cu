#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLessThanEq(float* mat1, float* mat2, float* target, unsigned int len) {
    // Calculate index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if index is within bounds
    if (idx < len) {
        // Direct element-wise comparison
        target[idx] = mat1[idx] <= mat2[idx];
    }
}