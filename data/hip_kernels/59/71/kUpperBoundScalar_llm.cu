#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kUpperBoundScalar(float* mat, float val, float* target, unsigned int len) {
    // Calculate the global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a stride loop with optimal increment
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        // Perform upper bound comparison and assignment
        target[i] = mat[i] > val ? val : mat[i];
    }
}