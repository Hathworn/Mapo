#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSetOnes(float *dest, int count){
for (int i = threadIdx.x; i < count; i += blockDim.x) {
dest[i] = 1;
}
}