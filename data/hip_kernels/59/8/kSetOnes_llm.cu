#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetOnes(float *dest, int count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    int stride = blockDim.x * gridDim.x;             // Calculate stride for grid stride loop

    for (int i = idx; i < count; i += stride) {
        dest[i] = 1;                                  // Set value to 1
    }
}