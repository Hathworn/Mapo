#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kRectifiedLinearDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll the loop for performance optimization
    for (unsigned int i = idx; i < numEls; i += blockDim.x * gridDim.x * 4) {
        if (i < numEls) dest[i] = a[i] * (b[i] > 0 ? 1 : 0);
        if (i + blockDim.x * gridDim.x < numEls) dest[i + blockDim.x * gridDim.x] = a[i + blockDim.x * gridDim.x] * (b[i + blockDim.x * gridDim.x] > 0 ? 1 : 0);
        if (i + 2 * blockDim.x * gridDim.x < numEls) dest[i + 2 * blockDim.x * gridDim.x] = a[i + 2 * blockDim.x * gridDim.x] * (b[i + 2 * blockDim.x * gridDim.x] > 0 ? 1 : 0);
        if (i + 3 * blockDim.x * gridDim.x < numEls) dest[i + 3 * blockDim.x * gridDim.x] = a[i + 3 * blockDim.x * gridDim.x] * (b[i + 3 * blockDim.x * gridDim.x] > 0 ? 1 : 0);
    }
}