#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultDiagonal(float* mat, float* vec, float* tgtMat, unsigned int width) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate the stride for coalesced memory access
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use shared memory for better performance
    extern __shared__ float s_vec[];

    // Preload vector data into shared memory
    if (idx < width) {
        s_vec[idx] = vec[idx];
    }
    __syncthreads();

    // Process diagonal elements using efficient stride-based loop
    for (unsigned int i = idx; i < width; i += stride) {
        tgtMat[width * i + i] = mat[width * i + i] * s_vec[i];
    }
}