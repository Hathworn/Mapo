#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kExtractPatches3(float* images, float* patches, float* width_offset, float* height_offset, float* flip, int num_images, int img_width, int img_height, int patch_width, int patch_height, int num_colors) {

    int dest_col = blockIdx.x * blockDim.x + threadIdx.x;
    int dest_row = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure we stay within patch dimensions
    if (dest_col < patch_width && dest_row < patch_height) {
        // Precompute some variables outside the loop
        int patch_size = patch_width * patch_height;
        int img_size = img_width * img_height * num_colors;

        for (unsigned int b = blockIdx.z; b < num_colors * num_images; b += gridDim.z) {
            int color = b % num_colors;
            int image_id = b / num_colors;

            // Calculate source row and column considering potential flip
            int source_row = int(height_offset[image_id]) + dest_row;
            int source_col = int(width_offset[image_id]) + dest_col;
            source_col = (flip[image_id] > 0.5) ? (img_width - source_col - 1) : source_col;

            // Compute linear indices for source and destination
            unsigned long source_index = source_col + img_width * (source_row + img_height * (color + num_colors * image_id));
            unsigned long dest_index = dest_col + patch_width * (dest_row + patch_height * (color + num_colors * image_id));

            // Remove unnecessary synchronization
            patches[dest_index] = images[source_index];
        }
    }
}