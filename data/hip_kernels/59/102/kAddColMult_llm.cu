#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddColMult(float* mat, float* vec, float* tgtMat, float mult, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;  // Use 'stride' instead of 'numThreads'

    for (unsigned int i = idx; i < width * height; i += stride) {
        unsigned int row = i % height;  // Calculate row once per iteration
        tgtMat[i] = mat[i] + mult * vec[row];
    }
}