#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kWriteRowsMult(float* data, float* target, int num_images, int num_modules, int num_modules_batch, int module_id_offset, float alpha, float beta) {
    int c = blockIdx.y;
    int src_module_id = blockIdx.x;
    int dst_module_id = module_id_offset + src_module_id;  // Remove redundant calculation

    data += num_images * (src_module_id + c * num_modules_batch);
    target += num_images * (dst_module_id + c * num_modules);

    int im = threadIdx.x;
    int stride = blockDim.x;  // Use stride for increment

    for (; im < num_images; im += stride) {
        target[im] = alpha * target[im] + beta * data[im];  // Use existing stride
    }
}