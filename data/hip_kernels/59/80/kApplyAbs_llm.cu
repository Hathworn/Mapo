#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kApplyAbs(float* mat, float* target, unsigned int len) {
    // Calculate global index using grid-stride loop for efficiency
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += blockDim.x * gridDim.x) {
        target[i] = fabsf(mat[i]); // Use built-in function for absolute value
    }
}