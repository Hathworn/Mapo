#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSoftMaxGrad(float* mat, float* labels, float* target, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x; // Use stride to access elements

    for (unsigned int i = idx; i < width * height; i += stride) {
        // Efficiently compute softmax gradient
        target[i] = mat[i] - ((int)labels[i / height] == i % height ? 1.0f : 0.0f);
    }
}