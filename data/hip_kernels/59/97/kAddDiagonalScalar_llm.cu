#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddDiagonalScalar(float* mat, float val, float* tgtMat, unsigned int width) {
    // Compute the unique thread index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Increment index by total number of threads to process all diagonal elements
    while (i < width) {
        tgtMat[width*i + i] = mat[width*i + i] + val; // Update diagonal element
        i += blockDim.x * gridDim.x;
    }
}