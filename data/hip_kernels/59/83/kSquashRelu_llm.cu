#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSquashRelu(float* mat, float* target, unsigned int len, float lambda) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Unroll the loop to optimize performance and reduce loop overhead
    for (unsigned int i = idx; i < len; i += stride) {
        float x = mat[i];
        float exp_val = __expf(-lambda * x);
        target[i] = 2 / (1 + exp_val) - 1; // Use precomputed exp_val
    }
}