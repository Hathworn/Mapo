#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kRectifyBoundingBox(float* boxes, float* width_offset, float* height_offset, float* flip, int num_images, int patch_width, int patch_height, int num_locs) {
    int loc_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (loc_id < num_locs) {
        float *xmin_block = boxes + num_images * loc_id,
        *ymin_block = boxes + num_images * (loc_id + num_locs),
        *xmax_block = boxes + num_images * (loc_id + num_locs * 2),
        *ymax_block = boxes + num_images * (loc_id + num_locs * 3);

        for (int image_id = threadIdx.y; image_id < num_images; image_id += blockDim.y) {
            float xmin = (flip[image_id] > 0.5f) ? (256.f / patch_width - xmax_block[image_id]) : xmin_block[image_id];
            float xmax = (flip[image_id] > 0.5f) ? (256.f / patch_width - xmin_block[image_id]) : xmax_block[image_id];
            float ymin = ymin_block[image_id];
            float ymax = ymax_block[image_id];
            float wo = width_offset[image_id];
            float ho = height_offset[image_id];

            // Precomputing division outside the loop
            float patch_width_inv = 1.f / patch_width;
            float patch_height_inv = 1.f / patch_height;

            xmin_block[image_id] = xmin - wo * patch_width_inv;
            xmax_block[image_id] = xmax - wo * patch_width_inv;
            ymin_block[image_id] = ymin - ho * patch_height_inv;
            ymax_block[image_id] = ymax - ho * patch_height_inv;
        }
    }
}