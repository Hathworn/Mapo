#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kReadRows(float* data, float* target, int num_images, int num_modules, int num_modules_batch, int module_id_offset) {
    // Calculate global thread index
    int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads_in_block = blockDim.x * gridDim.x;

    // Calculate the offsets
    int c = blockIdx.y;
    int src_module_id = module_id_offset + blockIdx.x;
    int dst_module_id = blockIdx.x;

    data += num_images * (src_module_id + c * num_modules);
    target += num_images * (dst_module_id + c * num_modules_batch);

    // Distribute workload across threads
    for (int im = global_thread_id; im < num_images; im += num_threads_in_block) {
        target[im] = data[im];
    }
}