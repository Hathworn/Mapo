#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCrossEntropy(float* mat, float* p, float* target, unsigned int len, float tiny) {
    // Cache frequently accessed values
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += stride) {
        // Perform computation and directly write to target
        float prob = p[i] + tiny;
        target[i] = -mat[i] * __logf(prob);
    }
}