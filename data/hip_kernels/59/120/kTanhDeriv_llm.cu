#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTanhDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEls) { // Ensure each thread only processes valid elements
        dest[idx] = a[idx] * (1.0 + b[idx]) * (1.0 - b[idx]);
    }
}