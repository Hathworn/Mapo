#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultScalar(float* mat, float alpha, float* dest, unsigned int len, float scale_targets) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Prefetch scale_targets condition outside the loop
    bool scale_cond = (scale_targets != 0);

    for (unsigned int i = idx; i < len; i += stride) {
        if (scale_cond) {
            dest[i] = scale_targets * dest[i] + alpha * mat[i];
        } else {
            dest[i] = alpha * mat[i];
        }
    }
}