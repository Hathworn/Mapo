#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSoftMaxCrossEntropy(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float tiny) {
    // Use shared memory for improved memory access efficiency
    extern __shared__ float sharedData[];
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    for (unsigned int i = idx; i < width; i += numThreads) {
        float label_value = labels[i];
        // Pre-fetch required data into shared memory
        float mat_value = mat[height * i + (int)label_value];
        target[i] = -__logf(mat_value + tiny);
    }
}