#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Compute linear index for the matrix
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Terminate threads that fall outside the matrix bounds
    if (idx >= width * height) return;
    // Compute row index and use it to add to the matrix element
    const unsigned int row = idx / width;
    tgtMat[idx] = mat[idx] + vec[row];
}