#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBesselRatioActivation(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use grid-stride loop for better performance.
    for (unsigned int i = idx; i < len; i += stride) {
        float r = mat[i];

        // Efficiently compute Bessel ratio and store result.
        float i1 = cyl_bessel_i1f(r);
        float i0 = cyl_bessel_i0f(r);
        target[i] = i1 / i0;
    }
}