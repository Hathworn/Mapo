#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplyLog1PlusExp(float* mat, float* target, unsigned int len) {
    // Calculate global index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for faster access to frequently used data
    __shared__ float sharedBuffer[256];

    // Load elements into shared memory
    if (idx < len) {
        sharedBuffer[threadIdx.x] = mat[idx];
    }

    __syncthreads();

    // Compute the result using shared memory
    if (idx < len) {
        float mat_i = sharedBuffer[threadIdx.x];
        if (mat_i > 0) 
            target[idx] = (__logf(1 + __expf(-mat_i)) + mat_i);
        else 
            target[idx] = __logf(1 + __expf(mat_i));
    }
}