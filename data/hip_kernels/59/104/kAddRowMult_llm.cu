#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel to add row-wise multiplied vector
__global__ void kAddRowMult(float* mat, float* vec, float* tgtMat, float mult, unsigned int width, unsigned int height) {
    // Calculate the thread index within the grid
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int rowIdx = idx / width; // Determine the row index
    const unsigned int colIdx = idx % width; // Determine the column index

    if (rowIdx < height) { // Ensure row index is within bounds
        tgtMat[idx] = mat[idx] + mult * vec[rowIdx];
    }
}