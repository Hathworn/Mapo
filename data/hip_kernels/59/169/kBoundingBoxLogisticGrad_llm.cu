#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBoundingBoxLogisticGrad(float* mat, int* bbox, int* label, int* seg, float* indices, float* width_offset, float* height_offset, int size, int width, int height, int depth, float scale_width, float scale_height, float* grad) {
    const int color = blockIdx.z;

    // Optimized: calculate only once per thread
    const int image_id = threadIdx.x;
    const int col = blockIdx.x;
    const int row = blockIdx.y;

    // Pre-check conditions before calculations
    if (col < width && row < height && image_id < size && color < depth) {
        int num_bboxes = 0, num_bboxes_of_this_depth = 0, num_bboxes_of_this_depth_inside = 0;
        const int src_image_id = (int)indices[image_id];
        const int src_col = (int)(scale_width * col);
        const int src_row = (int)(scale_height * row);

        const int start = seg[src_image_id];
        const int end = seg[src_image_id + 1];

        // Loop through bboxes once
        for (int box_id = start; box_id < end; box_id++) {
            const int l = label[box_id];
            const int x1 = bbox[box_id << 2] - width_offset[image_id];
            const int y1 = bbox[(box_id << 2) + 1] - height_offset[image_id];
            const int x2 = bbox[(box_id << 2) + 2] - width_offset[image_id];
            const int y2 = bbox[(box_id << 2) + 3] - height_offset[image_id];

            // Precalculate inside checking
            const int inside = (src_col >= x1 && src_col <= x2 && src_row >= y1 && src_row <= y2);

            num_bboxes += inside;
            num_bboxes_of_this_depth += (l == color);
            num_bboxes_of_this_depth_inside += (inside && (l == color));
        }
        
        const unsigned long i = image_id + size * (col + width * (row + height * color));

        // Simplified grad calculation
        if (num_bboxes > 0) {
            grad[i] = (num_bboxes_of_this_depth_inside > 0) ? (mat[i] - 1) : 0;
        } else {
            grad[i] = (num_bboxes_of_this_depth > 0) ? mat[i] : 0;
        }
    }
}