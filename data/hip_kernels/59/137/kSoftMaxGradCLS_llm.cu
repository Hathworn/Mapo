#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSoftMaxGradCLS(float* mat, int* labels, float* indices, float* target, unsigned int width, unsigned int height) {
    // Calculate the global index for the current thread
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Pre-compute number of elements
    const unsigned int total_elements = width * height;
    // Loop over the elements this thread is responsible for, with a stride length equal to total threads
    for (unsigned int i = idx; i < total_elements; i += blockDim.x * gridDim.x) {
        // Compute the row index and check the condition directly
        const unsigned int row = i / width;
        const int label = labels[(int)indices[row]];
        // Direct write to target with conditional assignment
        target[i] = mat[i] - (label == row ? 1.0f : 0.0f);
    }
}