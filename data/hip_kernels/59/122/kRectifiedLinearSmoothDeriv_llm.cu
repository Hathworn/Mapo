#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kRectifiedLinearSmoothDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize loop by ensuring each thread handles only the valid range
    if (idx < numEls) {
        dest[idx] = a[idx] * (1 - __expf(-b[idx]));
    }
}