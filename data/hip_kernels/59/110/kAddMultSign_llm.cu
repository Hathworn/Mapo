#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddMultSign(float* a, float* b, unsigned int numEls, float mult) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Optimize by replacing the loop with a stride pattern using blockDim.x
    for (unsigned int i = idx; i < numEls; i += blockDim.x * gridDim.x) {
        // Simplify ternary operation logic for efficiency
        a[i] += mult * ((b[i] > 0) - (b[i] < 0));
    }
}