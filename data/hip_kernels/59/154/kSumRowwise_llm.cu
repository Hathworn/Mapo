#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize: Use parallel reduction within each block to calculate row sums
__global__ void kSumRowwise(float* mat, float* target, unsigned int width, unsigned int height, float mult, float p) {
    extern __shared__ float sum_vals[];
    const int row = blockIdx.x;
    const int tid = threadIdx.x;

    if (row < height) {
        float sum = 0;
        float *data = mat + row;

        // Each thread processes multiple elements
        for (unsigned int i = tid; i < width; i += blockDim.x) {
            sum += data[i * height];
        }

        sum_vals[tid] = sum;
        __syncthreads();

        // Perform reduction to compute the total sum for the row
        for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2) {
            if (tid < stride) {
                sum_vals[tid] += sum_vals[tid + stride];
            }
            __syncthreads();
        }

        // The first thread in the block writes the result
        if (tid == 0) {
            target[row] = p * target[row] + mult * sum_vals[0];
        }
    }
}