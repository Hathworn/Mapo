#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Use blockIdx.y to allow threads to directly compute corresponding row operations
    const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < height && col < width) {
        const unsigned int i = row * width + col;
        tgtMat[i] = mat[i] + vec[row];
    }
}