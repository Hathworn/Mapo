#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLog(float* mat, float* target, unsigned int len, float tiny) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Simplified stride loop by handling boundary condition directly in a single pass
    if (idx < len) {
        target[idx] = __logf(mat[idx] + tiny);
    }
}