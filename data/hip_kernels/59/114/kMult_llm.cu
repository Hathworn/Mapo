#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMult(float* a, float* b, float* dest, unsigned int numEls, float scale_targets) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    
    for (unsigned int i = idx; i < numEls; i += numThreads) {
        // Use ternary operator to simplify logic
        dest[i] = (scale_targets == 0) ? (a[i] * b[i]) : (scale_targets * dest[i] + a[i] * b[i]);
    }
}