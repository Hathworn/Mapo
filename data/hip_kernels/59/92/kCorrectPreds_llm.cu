#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCorrectPreds(float* mat, float* p, float* target, unsigned int len, float cutoff) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    const unsigned int stride = blockDim.x * gridDim.x; // Total stride for all threads

    // Efficient loop with stride for memory access coalescing
    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = mat[i] * (p[i] >= cutoff) + (1 - mat[i]) * (p[i] < cutoff);
    }
}