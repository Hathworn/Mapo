#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kShuffleColumns(float* source, float* target, float* indices, int width, int height) {
    // Calculate the unique thread index, total thread count
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Loop through each available column for this thread
    for (unsigned int i = idx; i < height * ((width + 1) / 2); i += numThreads) {
        unsigned int column = 2 * (i / height);
        unsigned int row = i % height;

        // Check if column exceeds width, avoid out-of-bound access
        if (column + 1 < width) {
            unsigned int pos1 = height * (int)indices[column] + row;
            unsigned int pos2 = height * (int)indices[column + 1] + row;

            // Swap the source contents and assign to target positions
            float temp1 = source[pos1];
            float temp2 = source[pos2];
            target[pos1] = temp2;
            target[pos2] = temp1;
        } else {
            unsigned int pos1 = height * (int)indices[column] + row;
            target[pos1] = source[pos1];
        }
    }
}