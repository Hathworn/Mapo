#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSoftMaxGradRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Loop through the data with striding
    for (unsigned int i = idx; i < width * height; i += stride) {
        // Calculate target only once per thread
        unsigned int row = i / height;
        unsigned int col = i % height;
        target[i] = mat[i] - ((int)labels[col] == row ? 1 : 0);
    }
}