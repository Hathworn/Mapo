#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kUpperBoundModScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    // Optimize by simplifying loop stride calculation
    for (unsigned int i = idx; i < len; i += stride) {
        float m = mat[i];
        target[i] = m > val ? val : (m < -val ? -val : m);
    }
}