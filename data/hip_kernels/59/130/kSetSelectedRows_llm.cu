#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetSelectedRows(float* target, float* source, float* indices, int nRowIs, int nCols, int nTargetRows) {
    __shared__ int targetRowIndices[32];
    const int startSourceRowI = blockIdx.x * 32;
    const int tid = threadIdx.x;
    const int localNRowIs = min(32, nRowIs - startSourceRowI);

    // Load 32 row indices into shared memory
    if (tid < localNRowIs) {
        int tempIndex = int(indices[startSourceRowI + tid]);
        targetRowIndices[tid] = (tempIndex < 0) ? (tempIndex + nTargetRows) : tempIndex;
        if (targetRowIndices[tid] < 0 || targetRowIndices[tid] >= nTargetRows)
            targetRowIndices[tid] = -1;
    }
    __syncthreads();

    // Copy 32 rows using unrolled loops and shared memory for indices
    for (int i = 0; i < localNRowIs; i++) {
        const int sourceRowI = startSourceRowI + i;
        const int targetRowI = targetRowIndices[i];
        if (targetRowI != -1) {
            for (int colI = tid; colI < nCols; colI += 32) {
                target[targetRowI * nCols + colI] = source[sourceRowI * nCols + colI];
            }
        } else {
            for (int colI = tid; colI < nCols; colI += 32) {
                target[targetRowI * nCols + colI] = __int_as_float(0x7fffffff); // Set to NaN
            }
        }
    }
}