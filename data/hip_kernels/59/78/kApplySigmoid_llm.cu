#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float sigmoid(float x) {
    return 1.0f / (1.0f + __expf(-x));
}

__global__ void kApplySigmoid(float* __restrict__ mat, float* __restrict__ target, unsigned int len) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Optimize by eliminating unnecessary variable
    if (idx < len) {
        target[idx] = sigmoid(mat[idx]);
    }
}