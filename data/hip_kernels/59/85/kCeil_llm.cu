#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCeil(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {  // Process only if idx is within bounds
        target[idx] = ceil(mat[idx]);
    }
}