#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBlockify(float* source, float* target, int numdims, int blocksize) {
    const unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x; // Compute global index
    const unsigned int stride = blockDim.x * gridDim.x; // Compute stride for grid-stride loop

    // Grid-stride loop to optimize memory access and improve parallelism
    for (unsigned int target_ind = idx; target_ind < numdims; target_ind += stride) {
        const int block = target_ind / blocksize;
        target[target_ind] = source[block * blocksize + (blockIdx.x * numdims / blocksize)]; // Cache source index calculation
    }
}