#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLessThanScalar(float* mat, float val, float* target, unsigned int len) {
    // Use a single index calculation to improve code clarity and reduce potential errors
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check index bounds before updating target to avoid unnecessary operations
    if (idx < len) target[idx] = mat[idx] < val;
}