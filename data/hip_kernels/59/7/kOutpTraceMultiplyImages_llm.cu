#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kOutpTraceMultiplyImages(float *expanded_images, float *expanded_derivs, int num_images, int num_channels, int num_modules_batch, int kernel_size) {
    int color = blockIdx.y;
    int module_id = blockIdx.x;
    int thread_stride = blockDim.x;
    int expanded_images_offset = num_images * num_modules_batch * (kernel_size * color + module_id);
    int expanded_derivs_offset = num_images * num_modules_batch * color + num_images * module_id;

    expanded_images += expanded_images_offset;
    expanded_derivs += expanded_derivs_offset;

    // Optimize loop by reducing pointer arithmetic in the inner loop
    for (int kpos = 0; kpos < kernel_size; kpos++) {
        int kernel_offset = num_images * num_modules_batch * kpos;
        for (int im = threadIdx.x; im < num_images; im += thread_stride) {
            int image_idx = im + kernel_offset;
            expanded_images[image_idx] *= expanded_derivs[im];
        }
        // Synchronization is unnecessary between iterations since no dependencies
    }
}