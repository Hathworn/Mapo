#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDivide(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Bounds check; ensure index is within array limits
    if (idx < numEls) {
        dest[idx] = a[idx] / b[idx];
    }
}