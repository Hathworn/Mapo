#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLowerBound(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {  // Optimize with boundary check
        target[idx] = fmaxf(mat1[idx], mat2[idx]);  // Use fmaxf for performance
    }
}