#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAdd(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEls) { // Efficient conditional check
        dest[idx] = a[idx] + b[idx]; // Direct computation without loop
    }
}