#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDivByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate row and column indices for a 2D grid of blocks
    const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure that we do not go out of bounds
    if (col < width && row < height) {
        // Perform division operation
        tgtMat[row * width + col] = mat[row * width + col] / vec[row];
    }
}