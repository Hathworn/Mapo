#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kExtractPatches(float* images, float* patches, float* indices, float* width_offset, float* height_offset, int num_images, int img_width, int img_height, int patch_width, int patch_height, int num_colors) {
    const unsigned long idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned long numThreads = blockDim.x * gridDim.x;
    const unsigned long total_pixels = patch_width * patch_height * num_colors * num_images;

    for (unsigned long i = idx; i < total_pixels; i += numThreads) {
        unsigned long ind = i;
        unsigned long image_id = ind % num_images; 
        ind /= num_images;
        unsigned long dest_col = ind % patch_width; 
        ind /= patch_width;
        unsigned long dest_row = ind % patch_height; 
        ind /= patch_height;
        unsigned long color = ind % num_colors;

        unsigned long source_row = static_cast<unsigned long>(height_offset[image_id]) + dest_row;
        unsigned long source_col = static_cast<unsigned long>(width_offset[image_id]) + dest_col;
        // Use pre-calculated index offset to improve readability
        unsigned long pos = source_col + img_width * (source_row + img_height * (color + num_colors * static_cast<int>(indices[image_id])));
        patches[i] = images[pos];
    }
}