#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCrossEntropyBernoulli(float* mat, float* p, float* target, unsigned int len, float tiny) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    // Vectorize accesses by using the stride pattern
    for (unsigned int i = idx; i < len; i += stride) {
        float mat_val = mat[i];
        float p_val = p[i];
        
        // Reduce redundant memory accesses and multiplications
        float log_p = __logf(p_val + tiny);
        float log_one_minus_p = __logf(1 - p_val + tiny);
        
        // Replace decimal multiplications with pre-computed values
        target[i] = -mat_val * log_p - (1.0f - mat_val) * log_one_minus_p;
    }
}