#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBesselRatioActivationContinuedFraction(float* mat, float* target, float order, int num_terms, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x; // Use a stride to ensure better memory access pattern

    for (unsigned int i = idx; i < len; i += stride) {
        float k = mat[i];
        float reciprocal_k = 1.0f / k; // Precompute reciprocal of k for efficiency
        float result = 2.0f * (order + num_terms) * reciprocal_k;

        for (int j = num_terms - 1; j > 0; j--) {
            result = 2.0f * (order + j) * reciprocal_k + 1.0f / result;
        }

        target[i] = 1.0f / result;
    }
}