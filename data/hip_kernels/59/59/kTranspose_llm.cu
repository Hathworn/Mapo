#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTranspose(float *odata, float *idata, int width, int height) {
    __shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE + 1];

    // Calculate indices for input and output data
    unsigned int xIndex_in = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
    unsigned int yIndex_in = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;
    unsigned int xIndex_out = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
    unsigned int yIndex_out = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;

    // Load data into shared memory if within bounds
    if((xIndex_in < width) && (yIndex_in < height)) {
        unsigned int index_in = yIndex_in * width + xIndex_in;
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }

    __syncthreads();

    // Write transposed data to global memory if within bounds
    if((xIndex_out < height) && (yIndex_out < width)) {
        unsigned int index_out = yIndex_out * height + xIndex_out;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}