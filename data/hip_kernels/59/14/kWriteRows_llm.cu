#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kWriteRows(float* data, float* target, int num_images, int num_modules, int num_modules_batch, int module_id_offset, float beta) {
    int c = blockIdx.y;
    int src_module_id = blockIdx.x;
    int dst_module_id = module_id_offset + blockIdx.x;

    // Pre-calculate the offsets to avoid recalculating in the loop
    float* data_offset = data + num_images * (src_module_id + c * num_modules_batch);
    float* target_offset = target + num_images * (dst_module_id + c * num_modules);

    int im = threadIdx.x;
    int stride = blockDim.x;

    // Use a loop with stride to minimize thread divergence
    while (im < num_images) {
        target_offset[im] = beta * data_offset[im];
        im += stride;
    }
}