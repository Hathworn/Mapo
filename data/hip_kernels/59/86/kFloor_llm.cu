#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kFloor(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride to ensure all elements are processed and avoid diverging threads
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        target[i] = floor(mat[i]);  // Perform floor operation
    }
}