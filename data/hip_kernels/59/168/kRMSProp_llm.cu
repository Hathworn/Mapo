#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kRMSProp(float *history, float *grad, float factor, int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    // Compute frequently used value once per thread
    const float adjustedFactor = sqrt(factor);
    for (unsigned int i = idx; i < len; i += numThreads) {
        float h = history[i];
        float g = grad[i];
        // Simplified the computation by reducing repeated work
        history[i] = adjustedFactor * sqrt(h * h + ((1 - factor) / factor) * g * g);
    }
}