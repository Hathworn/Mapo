#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSinDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEls) { // Avoid out-of-bounds access
        dest[idx] = a[idx] * __cosf(b[idx]);
    }
}