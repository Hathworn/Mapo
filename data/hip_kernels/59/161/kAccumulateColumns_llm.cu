#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAccumulateColumns(float* mat, float* indices, float* target, int mat_width, int target_width, int height, float mult, int avg) {
    const int row = blockIdx.x + blockIdx.y * gridDim.x;
    const int column = threadIdx.x;

    // Ensure the current thread is within bounds
    if (row < height && column < target_width) {
        float cur_sum = 0.0f;
        unsigned int count = 0;

        // Unroll loop to improve memory access efficiency
        for (unsigned int i = 0; i < mat_width; i++) {
            int idx_match = (__float_as_int(indices[i]) == column);
            count += idx_match;
            cur_sum += idx_match * mat[row + i * height];
        }

        // Prevent division by zero and apply multiplication
        target[row + height * column] = mult * cur_sum / ((avg && count > 0) ? count : 1);
    }
}