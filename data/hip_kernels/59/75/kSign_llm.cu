#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSign(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Use stride loop with improved handling of synchronization and memory coalescing
    for (unsigned int i = idx; i < len; i += numThreads) {
        float value = mat[i];
        target[i] = (value > 0.0f) - (value < 0.0f); // More concise determination of sign
    }
}