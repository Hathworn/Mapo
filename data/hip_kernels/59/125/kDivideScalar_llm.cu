#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDivideScalar(float* mat, float alpha, float* dest, unsigned int len) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds before computation to avoid unnecessary loops
    if (idx < len) {
        // Perform the division directly
        dest[idx] = mat[idx] / alpha;
    }
}