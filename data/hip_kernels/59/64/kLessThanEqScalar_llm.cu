#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLessThanEqScalar(float* mat, float val, float* target, unsigned int len) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Iterate over elements with a stride equal to total number of threads
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        target[i] = mat[i] <= val; // Perform comparison and store result
    }
}