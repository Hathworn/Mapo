#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kHingeQuadraticRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float margin) {
    int image_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (image_id < height) {
        mat += image_id;
        target += image_id;
        
        const int correct_label = static_cast<int>(labels[image_id]);
        const float correct_label_score = mat[correct_label * height];
        
        float sum = 0;
        for (unsigned int i = 0; i < width; i++) {
            float diff = margin + mat[i * height] - correct_label_score;
            float grad = max(diff, 0.0f); // Use max to handle condition in a single line
            target[i * height] = (i == correct_label) ? 0.0f : grad;
            sum += (i == correct_label) ? 0.0f : grad; // Avoid redundant branches
        }
        target[correct_label * height] = -sum;
    }
}