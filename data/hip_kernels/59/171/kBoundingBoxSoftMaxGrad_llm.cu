#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBoundingBoxSoftMaxGrad(float* mat, int* bbox, int* label, int* seg, float* indices, float* width_offset, float* height_offset, int size, int width, int height, int depth, float scale_width, float scale_height, float* grad) {
    const unsigned int len = width * height * depth * size;
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    int image_id, src_image_id, start, end, box_id;
    float source_x, source_y;

    // Utilize shared memory for frequently accessed data
    __shared__ int sharedSeg[1024];
    __shared__ float sharedWidthOffset[1024];
    __shared__ float sharedHeightOffset[1024];

    if (threadIdx.x < size) {
        sharedSeg[threadIdx.x] = seg[threadIdx.x];
        sharedWidthOffset[threadIdx.x] = width_offset[threadIdx.x];
        sharedHeightOffset[threadIdx.x] = height_offset[threadIdx.x];
    }
    __syncthreads();

    for (unsigned int i = idx; i < len; i += numThreads) {
        int ind = i;
        image_id = ind % size; ind /= size;
        source_x = scale_width * (ind % width); ind /= width;
        source_y = scale_height * (ind % height); ind /= height;
        int source_depth = ind % depth;
        src_image_id = (int)indices[image_id];

        start = sharedSeg[src_image_id];
        end = sharedSeg[src_image_id + 1];
        int num_bboxes = 0;
        int num_bboxes_of_this_depth = 0;

        for (box_id = start; box_id < end; box_id++) {
            int x1 = bbox[box_id << 2] - sharedWidthOffset[image_id];
            int y1 = bbox[(box_id << 2) + 1] - sharedHeightOffset[image_id];
            int x2 = bbox[(box_id << 2) + 2] - sharedWidthOffset[image_id];
            int y2 = bbox[(box_id << 2) + 3] - sharedHeightOffset[image_id];

            int inside = (source_x >= x1 && source_x <= x2 && source_y >= y1 && source_y <= y2) ? 1 : 0;
            num_bboxes += inside;
            num_bboxes_of_this_depth += (inside == 1 && label[box_id] == source_depth) ? 1 : 0;
        }
        grad[i] = mat[i] - ((num_bboxes > 0) ? ((float)num_bboxes_of_this_depth / num_bboxes) : (source_depth == 0 ? 1 : 0));
    }
}