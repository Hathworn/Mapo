#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplyCos(float* mat, float* target, unsigned int len) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check idx before processing to avoid out of bounds access
    if (idx < len) {
        target[idx] = __cosf(mat[idx]);
    }
}