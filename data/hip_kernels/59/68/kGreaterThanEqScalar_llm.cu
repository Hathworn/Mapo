#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kGreaterThanEqScalar(float* mat, float val, float* target, unsigned int len) {
    // Calculate the unique thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread processes only if it is within the bounds
    if (idx < len) {
        target[idx] = mat[idx] >= val;
    }
}