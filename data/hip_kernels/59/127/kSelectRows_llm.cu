#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSelectRows(float* source, float* target, float* indices, int nRowIs, int nCols, int nSourceRows) {
    __shared__ int sourceRowIndices[32];
    const int startTargetRowI = blockIdx.x * 32;
    const int tid = threadIdx.x;
    const int localNRowIs = min(32, nRowIs - startTargetRowI);

    // Load row indices into shared memory by all threads
    if (tid < localNRowIs) {
        sourceRowIndices[tid] = int(indices[startTargetRowI + tid]);
        if (sourceRowIndices[tid] < 0)
            sourceRowIndices[tid] += nSourceRows;
        if (sourceRowIndices[tid] < 0 || sourceRowIndices[tid] >= nSourceRows)
            sourceRowIndices[tid] = -1;
    }
    __syncthreads();

    // Use a single loop to copy rows to improve performance
    for (int i = 0; i < localNRowIs; i++) {
        const int targetRowI = startTargetRowI + i;
        const int sourceRowI = sourceRowIndices[i];
        if (sourceRowI != -1) {  // Avoid copying if sourceRowI is invalid
            for (int colI = tid; colI < nCols; colI += 32) {
                target[targetRowI * nCols + colI] = source[sourceRowI * nCols + colI];
            }
        } else if (tid == 0) {  // Only the first thread sets invalid row to NaN
            for (int colI = 0; colI < nCols; colI++) {
                target[targetRowI * nCols + colI] = 1.0/0.0 - 1.0/0.0;
            }
        }
    }
}