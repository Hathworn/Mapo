#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogisticGrad(float* mat, float* targets, float* out_grad, unsigned int numEls) {
    // Calculate the global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check bounds and perform computation
    if (idx < numEls) {
        out_grad[idx] = (targets[idx] < 0) ? 0 : (mat[idx] - targets[idx]);
    }
}