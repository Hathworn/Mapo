#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kHingeLinearRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float margin) {
    int image_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (image_id < height) {
        mat += image_id;
        target += image_id;
        const int correct_label = (int)labels[image_id];
        const float correct_label_score = mat[correct_label * height];
        float sum = 0;

        // Loop unrolling for better parallel computation and reduced branching
        unsigned int i;
        for (i = 0; i < (width / 4) * 4; i += 4) {
            float diff0 = margin + mat[i * height] - correct_label_score;
            float diff1 = margin + mat[(i + 1) * height] - correct_label_score;
            float diff2 = margin + mat[(i + 2) * height] - correct_label_score;
            float diff3 = margin + mat[(i + 3) * height] - correct_label_score;

            float grad0 = (diff0 > 0) ? 1 : 0;
            float grad1 = (diff1 > 0) ? 1 : 0;
            float grad2 = (diff2 > 0) ? 1 : 0;
            float grad3 = (diff3 > 0) ? 1 : 0;

            target[i * height] = (i == correct_label) ? 0 : grad0;
            target[(i + 1) * height] = ((i + 1) == correct_label) ? 0 : grad1;
            target[(i + 2) * height] = ((i + 2) == correct_label) ? 0 : grad2;
            target[(i + 3) * height] = ((i + 3) == correct_label) ? 0 : grad3;

            sum += (i == correct_label) ? 0 : grad0;
            sum += ((i + 1) == correct_label) ? 0 : grad1;
            sum += ((i + 2) == correct_label) ? 0 : grad2;
            sum += ((i + 3) == correct_label) ? 0 : grad3;
        }

        // Handle remaining elements
        for (; i < width; i++) {
            float diff = margin + mat[i * height] - correct_label_score;
            float grad = (diff > 0) ? 1 : 0;
            target[i * height] = (i == correct_label) ? 0 : grad;
            sum += (i == correct_label) ? 0 : grad;
        }
        
        target[correct_label * height] = -sum;
    }
}