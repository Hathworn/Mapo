#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSoftMaxCrossEntropyRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float tiny) {
    // Calculate the global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x; // Use a variable for stride

    // Loop with stride for better memory access pattern
    for (unsigned int i = idx; i < height; i += stride) {
        int label_index = static_cast<int>(labels[i]); // Use static_cast for clarity and type safety
        target[i] = -__logf(mat[height * label_index + i] + tiny); // Precompute label index
    }
}