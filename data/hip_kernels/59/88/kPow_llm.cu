#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kPow(float* mat, float pow, float* target, unsigned int len) {
    // Calculate unique thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        target[idx] = powf(mat[idx], pow); // Perform operation only if within bounds
    }
}