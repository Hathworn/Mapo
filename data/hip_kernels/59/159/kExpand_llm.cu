#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kExpand(float* source, float* indices, float* target, int height, int width, int target_width) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Pre-compute total number of elements
    const unsigned int total = target_width * height;

    // Use a stride loop to allow for coalesced memory access
    for (unsigned int i = idx; i < total; i += blockDim.x * gridDim.x) {
        // Calculate position in source array
        const int pos = height * static_cast<int>(indices[i / height]) + (i % height);

        // Ensure valid memory access; else assign NaN
        target[i] = (pos < height * width) ? source[pos] : __hiloint2float(0x7fffffff, 0x7fffffff);
    }
}