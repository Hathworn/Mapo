#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAssignScalar(float* dest, float alpha, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a simplified range check with stride loop
    if (idx < len) {
        for (unsigned int i = idx; i < len; i += gridDim.x * blockDim.x) {
            dest[i] = alpha;
        }
    }
}