#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate row and column indices
    unsigned int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int colIdx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check bounds
    if (rowIdx < height && colIdx < width) {
        unsigned int idx = rowIdx * width + colIdx;
        // Multiply by corresponding row vector element
        tgtMat[idx] = mat[idx] * vec[rowIdx];
    }
}
```
