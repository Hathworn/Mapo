#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddScalar(float* a, float alpha, float* dest, unsigned int numEls) {
    // Calculate the global index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we stay within bounds and avoid unnecessary iterations
    if (idx < numEls) {
        dest[idx] = a[idx] + alpha;
    }
}