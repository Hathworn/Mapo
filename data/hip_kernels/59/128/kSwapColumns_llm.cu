#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSwapColumns(float* source, float* target, float* indices1, float* indices2, int cols, int width, int height) {
    __shared__ float sharedSource[BLOCK_SIZE]; // Use shared memory for faster access
    __shared__ float sharedTarget[BLOCK_SIZE]; // Use shared memory for faster access

    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    unsigned int column, row, source_pos, target_pos;

    for (unsigned int i = idx; i < height * cols; i += numThreads) {
        column = i / height;
        row = i % height;
        source_pos = height * (int)indices1[column] + row;
        target_pos = height * (int)indices2[column] + row;

        // Load elements into shared memory
        sharedSource[threadIdx.x] = source[source_pos];
        sharedTarget[threadIdx.x] = target[target_pos];
        
        __syncthreads(); // Ensure all threads have loaded their elements

        source[source_pos] = sharedTarget[threadIdx.x];
        target[target_pos] = sharedSource[threadIdx.x];
        
        __syncthreads(); // Ensure all swaps are complete before next iteration
    }
}