#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogisticDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEls) { // Ensure processing only within bounds
        dest[idx] = a[idx] * b[idx] * (1.0 - b[idx]); // Compute result for current index
    }
}