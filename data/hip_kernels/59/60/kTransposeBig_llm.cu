#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTransposeBig(float *odata, float *idata, int height, int width) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Precompute height x width to avoid repeated multiplication
    const unsigned int totalElements = width * height;

    for (unsigned int i = idx; i < totalElements; i += numThreads) {
        int r = i % width;
        int c = i / width;

        // Efficiently access memory using computed indices
        odata[i] = idata[height * r + c];
    }
}