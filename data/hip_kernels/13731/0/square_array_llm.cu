#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure idx is within bounds
    if (idx < N) {
        float val = a[idx];
        a[idx] = val * val;
    }
}