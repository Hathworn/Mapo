#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];

        // Use shared memory to reduce global memory access (if beneficial)
        __shared__ short2 sharedLoc[256]; // assuming blockDim.x <= 256
        sharedLoc[threadIdx.x] = loc;
        __syncthreads();

        // Scale and write back results using shared memory
        x[ptidx] = sharedLoc[threadIdx.x].x * scale;
        y[ptidx] = sharedLoc[threadIdx.x].y * scale;
    }
}