#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Helper function to extract the imaginary part of a float2 */
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

/* Optimized global kernel function */
__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x; // Compute column index
    int i_start = blockIdx.y * blockDim.y * w;     // Start position for row in block

    int totalThreads = blockDim.y * gridDim.y;     // Total threads in y dimension
    int stride = totalThreads * w;                 // Stride for all elements in a block row

    for (int i = i_start + threadIdx.y * w; i < w * h; i += stride)
    {
        if (j < w)
        {
            image[i + j] = value;                    // Set the value directly
        }
    }
}