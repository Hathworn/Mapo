#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate 1D thread index for coalesced memory access
    int idx = blockIdx.y * blockDim.y * s + threadIdx.y * s + blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= h * s) return; // Check bounds using 1D index

    float scale = normalization_factor[idx];
    float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);
    image[idx] *= invScale;
}