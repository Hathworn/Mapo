#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LinearPolynomForwardImpl(const float* probs, int batchSize, const float* values, int polynomCount, int outputDim, float* out) {
    // out: batch_elem0 dim0, dim1, dimk batch_elem1 dim0 dim1 dimk
    // threads are batch elements

    int polynomId = blockIdx.x;
    const int dimId = blockIdx.y;

    int tid = threadIdx.x;
    if (tid >= batchSize) {
        return;
    }

    float sum = 0;
    int probOffset = tid + polynomId * batchSize; // precompute offset for probs
    int valOffset = dimId + polynomId * outputDim; // precompute offset for values

    while (polynomId < polynomCount) {
        const float polynomProb = __ldg(probs + probOffset); // precomputed offset
        const float v = __ldg(values + valOffset); // precomputed offset
        sum += polynomProb * v;
        
        polynomId += gridDim.x;
        probOffset += gridDim.x * batchSize; // update offset
        valOffset += gridDim.x * outputDim; // update offset
    }

    atomicAdd(out + dimId * batchSize + tid, sum);
}