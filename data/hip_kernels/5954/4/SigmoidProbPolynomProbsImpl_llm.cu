#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SigmoidProbPolynomProbsImpl(
    const float* features, 
    int batchSize, 
    const int* splits, 
    const float* conditions, 
    const int* polynomOffsets, 
    int polynomCount, 
    float lambda, 
    float* probs
) {
    // Each thread computes the probabilities for its assigned feature set.
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if (threadId < batchSize) {
        features += threadId;
        probs += threadId;
        
        int polynomId = blockIdx.y;  // Use blockIdx.y to parallelize over polynomIDs
        while (polynomId < polynomCount) {
            int offset = polynomOffsets[polynomId];
            int nextOffset = polynomOffsets[polynomId + 1];
            const int depth = nextOffset - offset;

            float logProb = 0.0f;
            for (int i = 0; i < depth; ++i) {
                const int f = __ldg(splits + offset + i);
                const float c = __ldg(conditions + offset + i);
                const float x = __ldg(features + f * batchSize);
                const float val = -lambda * (x - c);
                const float expVal = 1.0f + expf(val);
                const float isTrueLogProb = isfinite(expVal) ? log(expVal) : val;
                logProb -= isTrueLogProb;
            }
            const float prob = expf(logProb);
            probs[polynomId * batchSize] = prob;
            polynomId += gridDim.y;  // Advance by block count in the y-dimension
        }
    }
}