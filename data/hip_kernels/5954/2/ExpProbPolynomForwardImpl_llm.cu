#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ExpProbPolynomForwardImpl(const float* probs, int batchSize, const float* values, int polynomCount, int outputDim, float* out) {

    // Simplify indices for better readability
    int polynomId = blockIdx.x;
    const int dimId = blockIdx.y;
    const int tid = threadIdx.x;

    if (tid >= batchSize) {
        return;
    }

    float sum = 0.0f;  // Initialize sum with 0.0f for precision
    const int batchOffset = tid;  // Calculate offsets for memory coalescing
    const int valueOffset = dimId; 

    // Optimize memory access and reduce repetitive calculations
    for (int pid = polynomId; pid < polynomCount; pid += gridDim.x) {
        const float polynomProb = __ldg(probs + pid * batchSize + batchOffset);
        const float polynomValue = __ldg(values + pid * outputDim + valueOffset);
        sum += polynomProb * polynomValue;
    }

    // Make atomic operations more readable
    atomicAdd(out + dimId * batchSize + batchOffset, sum);
}