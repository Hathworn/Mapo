#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LinearPolynomProbsImpl( const float* features, int batchSize, const int* splits, const float* conditions, const int* polynomOffsets, int polynomCount, float lambda, float* probs, const int* origFIds) {
// Each thread handles one element of the batch
int tid = threadIdx.x + blockIdx.x * blockDim.x;
if (tid < batchSize) {
    features += tid;
    probs += tid;

    for (int polynomId = threadIdx.y + blockIdx.y * blockDim.y; polynomId < polynomCount; polynomId += blockDim.y * gridDim.y) {
        int offset = polynomOffsets[polynomId];
        int nextOffset = polynomOffsets[polynomId + 1];
        const int depth = nextOffset - offset;
        const int origFId = origFIds[polynomId];

        bool zeroProb = false;
        for (int i = 0; i < depth; ++i) {
            if (zeroProb) {
                break; // Break loop early if probability is zero
            }

            const float c = __ldg(conditions + offset + i);
            const int f = __ldg(splits + offset + i);
            const float x = __ldg(features + f * batchSize);

            if (x <= c) {
                zeroProb = true; // Skip further checks if condition is met
            }
        }

        float prob = 0.0f;
        if (!zeroProb) {
            // Using original feature id for probability calculation
            prob = (origFId != -1) ? __ldg(features + origFId * batchSize) : 1.0f;
        }

        probs[polynomId * batchSize] = prob;
    }
}
}