#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ExpProbPolynomProbsImpl(const float* features, int batchSize, const int* splits, const float* conditions, const int* polynomOffsets, int polynomCount, float lambda, float* probs) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (threadId < batchSize) {
        features += threadId;
        probs += threadId;

        for (int polynomId = blockIdx.x; polynomId < polynomCount; polynomId += gridDim.x) {
            int offset = polynomOffsets[polynomId];
            int nextOffset = polynomOffsets[polynomId + 1];
            const int depth = nextOffset - offset;

            float logProb = 0;
            bool zeroProb = false;
            
            // Unroll the loop to enhance parallel computation (assuming depth is small)
            for (int i = 0; i < depth && !zeroProb; ++i) {
                const int f = __ldg(splits + offset + i);
                const float c = __ldg(conditions + offset + i);
                const float x = __ldg(features + f * batchSize);

                const float val = -lambda * x;
                const float expVal = 1.0f - expf(val);

                if (isfinite(log(expVal))) {
                    logProb += log(expVal);
                } else {
                    zeroProb = true;
                }
            }
            
            float prob = zeroProb ? 0.0f : expf(logProb);
            probs[polynomId * batchSize] = prob;
        }
    }
}