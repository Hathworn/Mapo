#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PolynomForwardImpl(const float* probs, int batchSize, const float* values, int polynomCount, int outputDim, float* out) {
    // Calculate polynomial and dimension indices
    int polynomId = blockIdx.x;
    const int dimId = blockIdx.y;
    int tid = threadIdx.x;

    // Early return if thread index is out of bounds
    if (tid >= batchSize) {
        return;
    }

    float sum = 0.0f;

    // Calculate starting pointers for this thread
    const float* probs_ptr = probs + tid;
    const float* values_ptr = values + dimId;

    // Unroll the loop manually for potential performance gain
    int step = gridDim.x; // Loop stride for polynomial index
    for (int i = polynomId; i < polynomCount; i += step) {
        const float polynomProb = __ldg(probs_ptr + i * batchSize);
        const float polyValue = __ldg(values_ptr + i * outputDim);
        sum += polynomProb * polyValue;
    }

    // Perform atomic add to accumulate results
    atomicAdd(out + dimId * batchSize + tid, sum);
}