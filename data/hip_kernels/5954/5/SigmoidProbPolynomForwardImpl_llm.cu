#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SigmoidProbPolynomForwardImpl(const float* probs, int batchSize, const float* values, int polynomCount, int outputDim, float* out) {
    // Each block works on a unique combination of polynomId and dimId
    int polynomId = blockIdx.x;
    const int dimId = blockIdx.y;
    int tid = threadIdx.x;

    if (tid >= batchSize) {
        return;
    }

    float sum = 0.0f;
    // Offset pointers for the current thread
    const float* probsPtr = probs + tid;
    const float* valuesPtr = values + dimId;

    // Loop through all polynomials
    while (polynomId < polynomCount) {
        // Cache read only data in registers for reuse
        const float polynomProb = __ldg(probsPtr + polynomId * batchSize);
        const float val = __ldg(valuesPtr + polynomId * outputDim);
        // Accumulate the polynomial probability multiplied by its corresponding value
        sum += polynomProb * val;
        polynomId += gridDim.x;
    }

    // Atomically update the output for this dimension and batch element
    atomicAdd(out + dimId * batchSize + tid, sum);
}