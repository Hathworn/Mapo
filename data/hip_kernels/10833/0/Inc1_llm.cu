#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc1(float *Ad, float *Bd){
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if(tx < 1 ){
        float localA = Ad[tx]; // Local variable for Ad[tx], reducing memory access
        for(int i = 0; i < ITER; i++){
            localA += 1.0f; // Use the local variable
            for(int j = 0; j < 256; j++){
                Bd[tx] = localA; // Use the result from the local variable
            }
        }
        Ad[tx] = localA; // Update Ad[tx] with the optimized result
    }
}