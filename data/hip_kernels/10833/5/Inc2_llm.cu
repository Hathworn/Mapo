#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Inc2(float *Ad, float *Bd) {
    // Use a more efficient block size
    int tx = hipThreadIdx_x + hipBlockIdx_x * hipBlockDim_x;
    if (tx < 1024) {
        float tmpA = Ad[tx]; // Cache Ad[tx] to a temporary variable
        for (int i = 0; i < ITER; i++) {
            tmpA += 1.0f;
            Ad[tx] = tmpA;
            float tmpB = tmpA; // Cache result for Bd[tx] to avoid redundant loads
            for (int j = 0; j < 256; j++) {
                Bd[tx] = tmpB; // Avoid recomputation inside the loop
            }
        }
    }
}