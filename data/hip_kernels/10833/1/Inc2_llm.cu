#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc2(float *Ad, float *Bd) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    // Use shared memory to minimize global memory access latency
    __shared__ float sharedA[256];
    
    if (tx < 1024) {
        // Load from global to shared memory
        float valueA = Ad[tx];
        for (int i = 0; i < ITER; i++) {
            // Perform computation using register
            valueA += 1.0f;
            // Reduce redundant computation by updating B only once per loop
            sharedA[threadIdx.x] = valueA;
        }
        Bd[tx] = sharedA[threadIdx.x];
        // Update global memory with the final value
        Ad[tx] = valueA;
    }
}