#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void clock_block(clock_t *d_o, clock_t clock_count)
{
    unsigned int start_clock = (unsigned int) clock();
    clock_t clock_offset = 0;

    // Reducing clock check frequency to minimize overhead
    while (clock_offset < clock_count)
    {
        unsigned int end_clock = (unsigned int) clock();
        clock_offset += end_clock - start_clock;
        start_clock = end_clock; // Avoid recomputing full offset each iteration
    }

    d_o[0] = clock_offset;
}