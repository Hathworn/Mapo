#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void Inc1(float *Ad, float *Bd) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation
    if(tx < 1) {
        float temp = Ad[tx]; // Store Ad[tx] in a temporary variable to reduce global memory access
        for(int i = 0; i < ITER; i++) {
            temp = temp + 1.0f; // Increment temporary variable
            for(int j = 0; j < 256; j++) {
                Bd[tx] = temp; // Only need to set Bd[tx] once due to loop with the same value
            }
        }
        Ad[tx] = temp; // Write the final result back to global memory
    }
}