#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global thread index for flattening
    int index = blockIdx.y * blockDim.y * w + threadIdx.y * w + blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check boundary before writing
    if (index >= h * w) return;
    
    image[index] = value;
}