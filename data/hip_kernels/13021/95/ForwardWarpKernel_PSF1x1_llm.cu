#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Check if the thread is out of bounds
    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Load u and v using strided memory access to enhance memory coalescing
    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Calculate target pixel's bottom left corner
    float cx = fmaf(u_, time_scale, static_cast<float>(j) + 1.0f);
    float cy = fmaf(v_, time_scale, static_cast<float>(i) + 1.0f);

    // Convert coordinates to nearest integer values using intrinsic function for better performance
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Read the source value
    float value = src[image_row_offset + j];

    // Update destination image if within bounds
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}