#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate point index
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (ptidx < npoints)
    {
        // Directly access memory with fewer loads
        x[ptidx] = loc_[ptidx].x * scale;
        y[ptidx] = loc_[ptidx].y * scale;
    }
}