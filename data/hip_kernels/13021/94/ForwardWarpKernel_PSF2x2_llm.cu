#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Compute the bottom-left corner of the target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py;
    float dx = modff (cx, &px);
    float dy = modff (cy, &py);
    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];

    // Common offset calculation
    int offset;
    float weight;

    // Bottom right corner
    offset = ty * image_stride + tx;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        weight = dx * dy;
        _atomicAdd (dst + offset, value * weight);
        _atomicAdd (normalization_factor + offset, weight);
    }

    // Bottom left corner
    tx -= 1;
    offset = ty * image_stride + tx;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        weight = (1.0f - dx) * dy;
        _atomicAdd (dst + offset, value * weight);
        _atomicAdd (normalization_factor + offset, weight);
    }

    // Upper left corner
    ty -= 1;
    offset = ty * image_stride + tx;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        weight = (1.0f - dx) * (1.0f - dy);
        _atomicAdd (dst + offset, value * weight);
        _atomicAdd (normalization_factor + offset, weight);
    }

    // Upper right corner
    tx += 1;
    offset = ty * image_stride + tx;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        weight = dx * (1.0f - dy);
        _atomicAdd (dst + offset, value * weight);
        _atomicAdd (normalization_factor + offset, weight);
    }
}