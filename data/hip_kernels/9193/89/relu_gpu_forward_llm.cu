#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, const float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Check and assign in a single operation
    if (tid < N) {
        float val = in[tid];
        out[tid] = fmaxf(val, 0.0f);  // Use fmaxf for better readability and potential performance
    }
}