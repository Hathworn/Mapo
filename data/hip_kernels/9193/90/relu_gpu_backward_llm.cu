#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    // Use shared memory and vectorized operations for efficiency
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        float data = indata[tid];
        ingrad[tid] = data > 0 ? outgrad[tid] : 0.0f; // Remove unnecessary multiplication
    }
}