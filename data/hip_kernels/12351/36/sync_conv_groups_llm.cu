#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use __syncthreads() to synchronize threads within a block
__global__ void sync_conv_groups() {
    // Simple synchronization point for all threads
    __syncthreads();
}