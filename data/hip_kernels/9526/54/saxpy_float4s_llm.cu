#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy_float4s ( float* y, float* x, float a, clock_t * timer_vals)
{
    // Optimize by precalculating loop invariant values
    unsigned int base_idx = blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x;
    unsigned int stride = COMPUTE_THREADS_PER_CTA * CTA_COUNT;

    float4 * x_as_float4 = (float4 *)x;
    float4 * y_as_float4 = (float4 *)y;

    for (int i=0; i < NUM_ITERS/4; i++) {      
        unsigned int idx = i * stride + base_idx;

        float4 tmp1_x = x_as_float4[idx];
        float4 tmp1_y = y_as_float4[idx];

        // Use float4 operations to reduce instructions
        y_as_float4[idx].x = a * tmp1_x.x + tmp1_y.x;
        y_as_float4[idx].y = a * tmp1_x.y + tmp1_y.y;
        y_as_float4[idx].z = a * tmp1_x.z + tmp1_y.z;
        y_as_float4[idx].w = a * tmp1_x.w + tmp1_y.w;
    }
}