#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemvn_kernel1_fermi(int n, int m, int n1, float alpha, float* A, int lda, float *x, float *y)
{
    const int ind = blockIdx.x * num_threads + threadIdx.x;

    // Check index before accessing memory
    if (ind < n) {
        A += ind;
        float res = 0.f;

        for(int i = 0; i < n1; i += sgemv_bs) {
            #pragma unroll
            for(int j = 0; j < sgemv_bs; j++) {
                res += A[0] * x[j];
                A += lda;
            }
            x += sgemv_bs;
        }
        
        // Write result only if within bounds
        y[ind] = alpha * res;
    }
}