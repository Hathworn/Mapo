#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy_baseline(float* y, float* x, float a, clock_t * timer_vals)
{
    // Precompute invariant expressions outside the loop
    unsigned int base_idx = blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x;
    unsigned int stride = COMPUTE_THREADS_PER_CTA * CTA_COUNT;

    for (int i = 0; i < NUM_ITERS; i++) {
        unsigned int idx = i * stride + base_idx;
        y[idx] = a * x[idx] + y[idx];
    }
}