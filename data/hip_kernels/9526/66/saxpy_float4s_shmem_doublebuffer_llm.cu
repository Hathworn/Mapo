#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy_float4s_shmem_doublebuffer (float* y, float* x, float a, clock_t * timer_vals) {
    __shared__ float4 sdata_x0_0[COMPUTE_THREADS_PER_CTA];
    __shared__ float4 sdata_y0_0[COMPUTE_THREADS_PER_CTA];
    __shared__ float4 sdata_x0_1[COMPUTE_THREADS_PER_CTA];
    __shared__ float4 sdata_y0_1[COMPUTE_THREADS_PER_CTA];

    int tid = threadIdx.x;
    unsigned int idx0 = blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;
    unsigned int idx1 = COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;

    float4 *x_as_float4 = (float4 *)x;
    float4 *y_as_float4 = (float4 *)y;
    float4 result_y;

    for (int i = 0; i < NUM_ITERS / 4; i += 2) {
        __syncthreads();
        float4 tmp1_x = x_as_float4[idx0];
        float4 tmp1_y = y_as_float4[idx0];

        if (i != 0) {
            result_y.x = a * sdata_x0_1[tid].x + sdata_y0_1[tid].x;
            result_y.y = a * sdata_x0_1[tid].y + sdata_y0_1[tid].y;
            result_y.z = a * sdata_x0_1[tid].z + sdata_y0_1[tid].z;
            result_y.w = a * sdata_x0_1[tid].w + sdata_y0_1[tid].w;
            y_as_float4[idx1] = result_y;
            idx1 += 2 * COMPUTE_THREADS_PER_CTA * CTA_COUNT;
        }

        sdata_x0_0[tid] = tmp1_x;
        sdata_y0_0[tid] = tmp1_y;
        __syncthreads();

        float4 tmp2_x = x_as_float4[idx1];
        float4 tmp2_y = y_as_float4[idx1];

        result_y.x = a * sdata_x0_0[tid].x + sdata_y0_0[tid].x;
        result_y.y = a * sdata_x0_0[tid].y + sdata_y0_0[tid].y;
        result_y.z = a * sdata_x0_0[tid].z + sdata_y0_0[tid].z;
        result_y.w = a * sdata_x0_0[tid].w + sdata_y0_0[tid].w;
        y_as_float4[idx0] = result_y;
        idx0 += 2 * COMPUTE_THREADS_PER_CTA * CTA_COUNT;

        sdata_x0_1[tid] = tmp2_x;
        sdata_y0_1[tid] = tmp2_y;
    }

    __syncthreads();
    result_y.x = a * sdata_x0_1[tid].x + sdata_y0_1[tid].x;
    result_y.y = a * sdata_x0_1[tid].y + sdata_y0_1[tid].y;
    result_y.z = a * sdata_x0_1[tid].z + sdata_y0_1[tid].z;
    result_y.w = a * sdata_x0_1[tid].w + sdata_y0_1[tid].w;
    y_as_float4[idx1] = result_y;
}