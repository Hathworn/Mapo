#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy_float4s_shmem(float* y, float* x, float a, clock_t * timer_vals)
{
    extern __shared__ float sdata[]; // Use a single shared memory array
    float* sdata_x = sdata; // Pointer for X shared memory
    float* sdata_y = sdata + COMPUTE_THREADS_PER_CTA * 4; // Pointer for Y shared memory

    int tid = threadIdx.x;

    for (int i = 0; i < NUM_ITERS / 4; i++)
    {
        unsigned int idx = i * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x;

        __syncthreads();
        float4* x_as_float4 = (float4*)x;
        float4* y_as_float4 = (float4*)y;
        float4 tmp1_x, tmp1_y;
        tmp1_x = x_as_float4[idx];
        tmp1_y = y_as_float4[idx];

        // Use a single array for shared memory
        sdata_x[tid] = tmp1_x.x;
        sdata_x[tid + COMPUTE_THREADS_PER_CTA] = tmp1_x.y;
        sdata_x[tid + 2 * COMPUTE_THREADS_PER_CTA] = tmp1_x.z;
        sdata_x[tid + 3 * COMPUTE_THREADS_PER_CTA] = tmp1_x.w;
        sdata_y[tid] = tmp1_y.x;
        sdata_y[tid + COMPUTE_THREADS_PER_CTA] = tmp1_y.y;
        sdata_y[tid + 2 * COMPUTE_THREADS_PER_CTA] = tmp1_y.z;
        sdata_y[tid + 3 * COMPUTE_THREADS_PER_CTA] = tmp1_y.w;
        __syncthreads();

        float4 result_y;
        result_y.x = a * sdata_x[tid] + sdata_y[tid];
        result_y.y = a * sdata_x[tid + COMPUTE_THREADS_PER_CTA] + sdata_y[tid + COMPUTE_THREADS_PER_CTA];
        result_y.z = a * sdata_x[tid + 2 * COMPUTE_THREADS_PER_CTA] + sdata_y[tid + 2 * COMPUTE_THREADS_PER_CTA];
        result_y.w = a * sdata_x[tid + 3 * COMPUTE_THREADS_PER_CTA] + sdata_y[tid + 3 * COMPUTE_THREADS_PER_CTA];
        y_as_float4[idx] = result_y;
    }
}