#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy_float4s (float* y, float* x, float a, clock_t * timer_vals)
{
    // Calculate the start index for this thread
    unsigned int start_idx = (blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x) * (NUM_ITERS / 4);
    float4 *x_as_float4 = (float4 *)x;
    float4 *y_as_float4 = (float4 *)y;

    // Loop unrolling to exploit more ILP
    for (int i = 0; i < (NUM_ITERS / 4); i += 4) {
        #pragma unroll 4
        for (int j = 0; j < 4; ++j) {
            unsigned int idx = start_idx + i + j;
            float4 tmp_x = x_as_float4[idx];
            float4 tmp_y = y_as_float4[idx];

            // Compute result using fused multiply-add for better performance
            tmp_y.x = fmaf(a, tmp_x.x, tmp_y.x);
            tmp_y.y = fmaf(a, tmp_x.y, tmp_y.y);
            tmp_y.z = fmaf(a, tmp_x.z, tmp_y.z);
            tmp_y.w = fmaf(a, tmp_x.w, tmp_y.w);

            y_as_float4[idx] = tmp_y;
        }
    }
}