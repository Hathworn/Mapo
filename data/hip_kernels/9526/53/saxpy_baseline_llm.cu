#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy_optimized(float* y, float* x, float a, clock_t* timer_vals)
{
  // Pre-compute index offset to avoid repeated calculation inside the loop
  unsigned int base_idx = blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x;
  for (int i = 0; i < NUM_ITERS; i++) {
    unsigned int idx = i * COMPUTE_THREADS_PER_CTA * CTA_COUNT + base_idx;
    y[idx] = a * x[idx] + y[idx];
  }
}