#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemvn_kernel2_fermi(int n, int m, int n1, float alpha, float* A, int lda, float *x, float *y) {
    int ind = blockIdx.x * num_threads + threadIdx.x;

    A += ind;
    x += threadIdx.x;

    float res = 0.f;

    __shared__ float buff[num_threads];
    for (int i = 0; i < n1; i += num_threads) {
        // Load elements into shared memory
        __syncthreads();
        buff[threadIdx.x] = x[i];
        __syncthreads();

        // Compute results
        #pragma unroll
        for (int j = 0; j < num_threads; j++) {
            res += A[0] * buff[j];
            A += lda;
        }
    }

    // Final result storage
    if (ind < n) {
        y[ind] = alpha * res;
    }
}
```
