#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy_shmem_doublebuffer ( float* y, float* x, float a, clock_t * timer_vals)
{
    volatile __shared__ float sdata_x0[COMPUTE_THREADS_PER_CTA];
    volatile __shared__ float sdata_y0[COMPUTE_THREADS_PER_CTA];
    volatile __shared__ float sdata_x1[COMPUTE_THREADS_PER_CTA];
    volatile __shared__ float sdata_y1[COMPUTE_THREADS_PER_CTA];
    int tid = threadIdx.x;
    unsigned int idx0 = blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;
    unsigned int idx1 = COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;
    // Preload data into shared memory to minimize bank conflicts
    sdata_x0[tid] = x[idx0];
    sdata_y0[tid] = y[idx0];
    __syncthreads();
    for (int i = 0; i < NUM_ITERS; i += 2) {
        // Use shared memory to reduce global memory access latency
        if (i != 0) {
            y[idx1] = a * sdata_x1[tid] + sdata_y1[tid];
            idx1 += 2 * COMPUTE_THREADS_PER_CTA * CTA_COUNT;
        }
        __syncthreads();
        sdata_x1[tid] = x[idx1];
        sdata_y1[tid] = y[idx1];
        y[idx0] = a * sdata_x0[tid] + sdata_y0[tid];
        idx0 += 2 * COMPUTE_THREADS_PER_CTA * CTA_COUNT;
        __syncthreads();
        sdata_x0[tid] = x[idx0];
        sdata_y0[tid] = y[idx0];
        __syncthreads();
    }
    y[idx1] = a * sdata_x1[tid] + sdata_y1[tid];
}