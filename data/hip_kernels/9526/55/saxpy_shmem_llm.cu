#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy_shmem(float* y, float* x, float a, clock_t * timer_vals) {
    extern __shared__ float sdata[]; // Use a single shared memory allocation
    float* sdata_x0 = sdata;
    float* sdata_y0 = sdata + COMPUTE_THREADS_PER_CTA;
    
    int tid = threadIdx.x;
    int base_idx = blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;
    
    for (int i = 0; i < NUM_ITERS; ++i) {
        unsigned int idx = i * COMPUTE_THREADS_PER_CTA * CTA_COUNT + base_idx;
        
        // Load x and y values into shared memory
        sdata_x0[tid] = x[idx];
        sdata_y0[tid] = y[idx];
        __syncthreads();
        
        // Compute y using shared memory values
        y[idx] = a * sdata_x0[tid] + sdata_y0[tid];
        __syncthreads(); // Ensure all threads have completed
    }
}