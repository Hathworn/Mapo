#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy_float4s_shmem (float* y, float* x, float a, clock_t *timer_vals)
{
    // Use float4 shared memory to optimize access pattern
    volatile __shared__ float4 sdata_x[COMPUTE_THREADS_PER_CTA];
    volatile __shared__ float4 sdata_y[COMPUTE_THREADS_PER_CTA];
    int tid = threadIdx.x;

    for (int i = 0; i < NUM_ITERS / 4; i++) {
        unsigned int idx = i * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x;

        __syncthreads(); // Make sure shared memory is ready
        float4 *x_as_float4 = (float4 *)x;
        float4 *y_as_float4 = (float4 *)y;
        
        // Coalesced memory access
        sdata_x[tid] = x_as_float4[idx];
        sdata_y[tid] = y_as_float4[idx];
        
        __syncthreads(); // Ensure all threads have loaded their data

        // Perform SAXPY operation
        float4 result_y;
        result_y.x = a * sdata_x[tid].x + sdata_y[tid].x;
        result_y.y = a * sdata_x[tid].y + sdata_y[tid].y;
        result_y.z = a * sdata_x[tid].z + sdata_y[tid].z;
        result_y.w = a * sdata_x[tid].w + sdata_y[tid].w;

        // Coalesced memory write
        y_as_float4[idx] = result_y;
    }
}