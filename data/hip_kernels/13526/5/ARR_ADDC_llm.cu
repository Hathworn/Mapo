#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ARR_ADDC(float* __restrict__ result, const float* __restrict__ in1, const float* __restrict__ in2, int N)
{
    // Calculate global thread index
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure index is within bounds
    if (index < N)
    {
        // Perform addition
        result[index] = in1[index] + in2[index];
    }
}