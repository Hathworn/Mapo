#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate the fractional and integer position
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);

    float value = src[image_row_offset + j];
    float weight;

    // Shared memory for atomic operations (Optional for further optimization)
    extern __shared__ float shared_memory[];
    float *shared_dst = shared_memory;
    float *shared_norm = shared_memory + blockDim.x * blockDim.y;

    // Fill pixel containing bottom right corner
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        weight = dx * dy;
        atomicAdd(&shared_dst[ty * image_stride + tx], value * weight);
        atomicAdd(&shared_norm[ty * image_stride + tx], weight);
    }

    // Fill pixel containing bottom left corner
    tx -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        weight = (1.0f - dx) * dy;
        atomicAdd(&shared_dst[ty * image_stride + tx], value * weight);
        atomicAdd(&shared_norm[ty * image_stride + tx], weight);
    }

    // Fill pixel containing upper left corner
    ty -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(&shared_dst[ty * image_stride + tx], value * weight);
        atomicAdd(&shared_norm[ty * image_stride + tx], weight);
    }

    // Fill pixel containing upper right corner
    tx += 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        weight = dx * (1.0f - dy);
        atomicAdd(&shared_dst[ty * image_stride + tx], value * weight);
        atomicAdd(&shared_norm[ty * image_stride + tx], weight);
    }

    // Write back to global memory
    __syncthreads();
    if (threadIdx.x < blockDim.x && threadIdx.y < blockDim.y) 
    {
        int global_index = (blockIdx.y * blockDim.y + threadIdx.y) * image_stride 
                           + (blockIdx.x * blockDim.x + threadIdx.x);
        if (global_index < w * h)
        {
            atomicAdd(&dst[global_index], shared_dst[threadIdx.y * blockDim.x + threadIdx.x]);
            atomicAdd(&normalization_factor[global_index], shared_norm[threadIdx.y * blockDim.x + threadIdx.x]);
        }
    }
}