#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global thread index
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within bounds
    if (i < h && j < w) 
    {
        const int pos = i * s + j;

        float scale = normalization_factor[pos];

        // Perform normalization with guard for division by zero
        image[pos] *= (scale == 0.0f) ? 1.0f : (1.0f / scale);
    }
}