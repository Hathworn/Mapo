#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* __restrict__ loc_, float* __restrict__ x, float* __restrict__ y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index is within bounds
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];

        // Perform scaled assignment
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}