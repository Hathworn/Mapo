#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Calculate target coordinates using time scale and current indices
    float cx = fmaf(u_, time_scale, float(j) + 1.0f); // fused multiply-add for efficiency
    float cy = fmaf(v_, time_scale, float(i) + 1.0f);

    // Pixel containing bottom left corner
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    float value = src[image_row_offset + j];

    // Check pixel boundary conditions and update destination
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) 
    {
        atomicAdd(dst + ty * image_stride + tx, value); // Directly use atomicAdd for thread safety
    }
}