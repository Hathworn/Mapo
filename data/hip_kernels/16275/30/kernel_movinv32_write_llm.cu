#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    // Obtain global thread index
    unsigned int idx = blockIdx.x * BLOCKSIZE + threadIdx.x * sizeof(unsigned int);
    
    // Ensure we don't exceed the end_ptr
    if (idx >= (unsigned int*)end_ptr - (unsigned int*)_ptr) {
        return;
    }

    // Calculate starting pointer for this thread
    unsigned int* ptr = (unsigned int*)(_ptr + idx);

    // Initialize pat and k
    unsigned int k = offset + threadIdx.x * (BLOCKSIZE / sizeof(unsigned int));
    unsigned int pat = pattern;

    // Loop unrolling to improve performance
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        ptr[i] = pat;
        k++;
        if (k >= 32) {
            k = 0;
            pat = lb;
        } else {
            pat = pat << 1;
            pat |= sval;
        }
    }
}