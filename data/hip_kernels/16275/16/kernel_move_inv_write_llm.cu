#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Exit early if outside the valid range
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Calculate the number of iterations outside the loop
    unsigned int num_iterations = BLOCKSIZE / sizeof(unsigned int);
    
    // Loop unrolling for better performance
    for (unsigned int i = 0; i < num_iterations; i += 4) {
        ptr[i] = pattern;
        if (i + 1 < num_iterations) ptr[i + 1] = pattern;
        if (i + 2 < num_iterations) ptr[i + 2] = pattern;
        if (i + 3 < num_iterations) ptr[i + 3] = pattern;
    }

    return;
}