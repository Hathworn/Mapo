#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    // Calculate the start pointer for the current block
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Return early if the start pointer exceeds the end pointer
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Caching the end index and step size to avoid recalculating
    unsigned int end_index = BLOCKSIZE / sizeof(unsigned int);

    // Unrolling loop for better performance
    for (unsigned int i = offset; i < end_index; i += MOD_SZ) {
        ptr[i] = p1;
    }

    // Unrolling loop for better performance
    for (unsigned int i = 0; i < end_index; i++) {
        if (i % MOD_SZ != offset) {
            ptr[i] = p2;
        }
    }
}