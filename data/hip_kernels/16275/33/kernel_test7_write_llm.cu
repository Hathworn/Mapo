#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    unsigned int idx = blockIdx.x * BLOCKSIZE / sizeof(unsigned int);
    unsigned int* ptr = (unsigned int*)_ptr + idx;
    unsigned int* start_ptr = (unsigned int*)_start_ptr;

    // Check ptr boundary condition
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Use threadIdx.x to perform coalesced writes
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        ptr[i] = start_ptr[i];
    }
}