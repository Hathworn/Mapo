#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_init(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int p1 = 1;
    // Unrolling the loop for better performance by reducing loop control overhead
    for (int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i += 16) {
        unsigned int p2 = ~p1;
        #pragma unroll
        for (int j = 0; j < 16; j += 4) {
            ptr[i + j] = ptr[i + j + 1] = p1;
            ptr[i + j + 2] = ptr[i + j + 3] = p2;
        }
        p1 = (p1 << 1) | (p1 == 0);
    }

    return;
}