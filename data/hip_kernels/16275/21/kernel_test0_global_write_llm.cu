#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int* orig_ptr = ptr;
    unsigned int pattern = 1;
    unsigned long mask = 4;

    *ptr = pattern;  // Set initial pattern

    while (true) {  // Optimize loop and control flow
        ptr = (unsigned int*)((unsigned long)orig_ptr | mask);
        
        if (ptr >= end_ptr) {  
            break;  // Exit loop if ptr exceeds end_ptr
        }

        *ptr = pattern;  // Write pattern
        pattern <<= 1;   // Update pattern
        mask <<= 1;      // Update mask
    }
    return;
}