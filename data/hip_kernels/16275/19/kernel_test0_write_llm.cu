#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    // Calculate start pointer for each block
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Exit if block is out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }
    
    // Use threadIdx.x in loop to allow parallel writing by individual threads within a block
    unsigned int idx = threadIdx.x;
    
    // Number of elements each thread writes
    unsigned int elems_per_thread = (BLOCKSIZE / sizeof(unsigned int)) / blockDim.x;
    
    // Optimization: Unroll loop for better performance
    #pragma unroll
    for (unsigned int j = 0; j < elems_per_thread; j++) {
        ptr[idx * elems_per_thread + j] = pattern;
    }

    return;
}