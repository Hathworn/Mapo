#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    // Calculate index using threadIdx.x for better utilization of threads
    unsigned long* ptr = (unsigned long*)(_ptr + (blockIdx.x * blockDim.x + threadIdx.x) * sizeof(unsigned long));

    // Early exit if pointer is beyond the end
    if (ptr >= (unsigned long*) end_ptr) {
        return;
    }

    // Use blockDim.x to efficiently access and write to memory
    unsigned int stride = blockDim.x * gridDim.x;
    for (unsigned long* p = ptr; p < (unsigned long*)end_ptr; p += stride) {
        *p = (unsigned long)p;
    }
}