#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*------------------------GPU RANKING----------------------------------------START-------*/

/*------------------------shfl_scan_test-----------------------------------------------Start*/
/*------------------------shfl_scan_test-----------------------------------------------End*/
/*------------------------Final Ranking-----------------------------------------------Start*/

/*------------------------Final_ranking-----------------------------------------------End*/

/*-----------------------GPU RANKING------------------------------------------END--------*/

/*-----------------------iDivUp--------------------------------------------------------Start*/

__global__ void final_ranking(float *data, int *rank, float *partial_data, int *partial_rank, int len) {
    extern __shared__ float shared_buf[]; // Use shared memory dynamically
    float &value_buf = shared_buf[0]; // First element for value
    int &rank_buf = *((int*) &shared_buf[1]); // Second element for rank

    int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
    if (id >= len) return; // Correct condition for valid index

    if (threadIdx.x == 0) {
        value_buf = partial_data[blockIdx.x];
        rank_buf = partial_rank[blockIdx.x];
    }
    __syncthreads(); // Ensure all threads see the updated values

    if (data[id] == value_buf) {
        rank[id] = rank_buf;
    }
}