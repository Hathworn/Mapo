#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void check_collisions( float x1_robot, float y1_robot, float x2_robot, float y2_robot, float *x1_obs, float *y1_obs, float *x2_obs, float *y2_obs, bool *collisions, int *indexes)
{
    int obstacleId = blockIdx.x * blockDim.x + threadIdx.x; // Use global indexing for scalability

    // Access values only once for memory efficiency
    float x1o = x1_obs[obstacleId];
    float y1o = y1_obs[obstacleId];
    float x2o = x2_obs[obstacleId];
    float y2o = y2_obs[obstacleId];

    // Calculate collision using pre-loaded values
    bool xcol = ((x1o <= x1_robot && x1_robot <= x2o) || (x1o <= x2_robot && x2_robot <= x2o)) || (x1_robot <= x1o && x2_robot >= x2o);
    bool ycol = ((y1o <= y1_robot && y1_robot <= y2o) || (y1o <= y2_robot && y2_robot <= y2o)) || (y1_robot <= y1o && y2_robot >= y2o);

    collisions[obstacleId] = (xcol && ycol);
}