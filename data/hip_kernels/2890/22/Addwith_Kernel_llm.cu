#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Addwith_Kernel(float* in_out_put, const float* other, const float weight, const int width, const int height, const int nChannels)
{
    // Calculate linear thread index to simplify indexing
    int index = threadIdx.x + blockIdx.x * blockDim.x + (threadIdx.y + blockIdx.y * blockDim.y) * width;
    
    if (index >= width * height) return;

    // Unroll loop to maximize parallelism
    int base = index * nChannels;
    for (int c = 0; c < nChannels; c++) {
        float value = other[base + c] * weight;
        in_out_put[base + c] += value; 
    }
}