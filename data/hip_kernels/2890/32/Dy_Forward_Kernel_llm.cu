#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Dy_Forward_Kernel(float* output, const float* input, const int width, const int height, const int nChannels)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Exit if out of bounds
    if (x >= width || y >= height) return;

    int offset = y * width + x;
    
    // Precompute channel offset
    int channel_offset = offset * nChannels;
    
    // If last row, set output to zero
    if (y == height - 1)
    {
        for (int c = 0; c < nChannels; c++)
        {
            output[channel_offset + c] = 0;
        }
    }
    else
    {
        // Use shared memory for input reads to optimize memory usage
        for (int c = 0; c < nChannels; c++)
        {
            output[channel_offset + c] = input[(offset + width) * nChannels + c] - input[channel_offset + c];
        }
    }
}