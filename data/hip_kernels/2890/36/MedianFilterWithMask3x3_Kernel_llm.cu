#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MedianFilterWithMask3x3_Kernel(float* output, const float* input, const int width, const int height, const int nChannels, const bool* keep_mask)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= width || y >= height)
        return;

    int offset = y * width + x;

    if (keep_mask[offset])
    {
        for (int c = 0; c < nChannels; c++)
            output[offset * nChannels + c] = input[offset * nChannels + c];
        return;
    }

    float vals[9];
    int count;
    int start_x = max(x - 1, 0);  // Determine start x with boundary check
    int end_x = min(x + 1, width - 1);  // Determine end x with boundary check
    int start_y = max(y - 1, 0);  // Determine start y with boundary check
    int end_y = min(y + 1, height - 1);  // Determine end y with boundary check

    for (int c = 0; c < nChannels; c++)
    {
        count = 0;
        for (int ii = start_y; ii <= end_y; ii++)
        {
            for (int jj = start_x; jj <= end_x; jj++)
            {
                int cur_offset = ii * width + jj;
                if (keep_mask[cur_offset])
                {
                    vals[count++] = input[cur_offset * nChannels + c];
                }
            }
        }

        if (count == 0)
        {
            output[offset * nChannels + c] = 0;
        }
        else
        {
            int mid = count / 2;  // Find the median index
            for (int pass = 0; pass <= mid; pass++)
            {
                int min_id = pass;
                for (int id = pass + 1; id < count; id++)
                {
                    if (vals[id] < vals[min_id])
                    {
                        min_id = id;
                    }
                }
                float temp = vals[pass];
                vals[pass] = vals[min_id];
                vals[min_id] = temp;  // Swap to sort for median
            }
            output[offset * nChannels + c] = vals[mid];
        }
    }
}