#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Compute_weightx_weighty1_Kernel(float* weightx, float* weighty, const float* psi, const float* phi, const float* absIx, const float* absIy, int nPixels, float norm_for_smooth_term, float eps)
{
    // Calculate the unique thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= nPixels) return;  // Terminate the thread if out of bounds

    float absIx_val = absIx[x];
    float absIy_val = absIy[x];
    float psi_val = psi[x];
    float phi_val = phi[x];

    // Precompute values based on norm_for_smooth_term
    if (norm_for_smooth_term == 2) {
        weightx[x] = psi_val;
        weighty[x] = phi_val;
    } else {
        float denom_x, denom_y;
        if (norm_for_smooth_term == 1) {  // L1 norm
            denom_x = absIx_val + eps;
            denom_y = absIy_val + eps;
        } else if (norm_for_smooth_term == 0) {  // Arbitrary small norm
            denom_x = absIx_val * absIx_val + eps;
            denom_y = absIy_val * absIy_val + eps;
        } else {  // General case
            float power = 2.0f - norm_for_smooth_term;
            denom_x = pow(absIx_val, power) + eps;
            denom_y = pow(absIy_val, power) + eps;
        }

        // Update weights
        weightx[x] = psi_val / denom_x;
        weighty[x] = phi_val / denom_y;
    }
}