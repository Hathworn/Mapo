#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Solve_redblack1_new_Kernel(float* output, const float* input, int width, int height, int nChannels, int c, const float* weightdata, const float* weightx, const float* weighty, float lambda, float omega, bool redflag)
{
    // Calculate global thread x and y indices
    int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2 + (redflag ? threadIdx.y % 2 : (threadIdx.y % 2 + 1));
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check for out-of-bound indices
    if (x >= width || y >= height)
        return;

    int offset = y * width + x;
    int slice = width * nChannels;
    int offset_c = offset * nChannels + c;
    
    // Initialize coefficients and sigma
    float coeff = weightdata[offset];
    float sigma = coeff * input[offset_c];

    // Compute contributions from neighboring pixels
    if (y > 0) {
        float weight = lambda * weighty[offset - width];
        coeff += weight;
        sigma += weight * output[offset_c - slice];
    }
    if (y < height - 1) {
        float weight = lambda * weighty[offset];
        coeff += weight;
        sigma += weight * output[offset_c + slice];
    }
    if (x > 0) {
        float weight = lambda * weightx[offset - 1];
        coeff += weight;
        sigma += weight * output[offset_c - nChannels];
    }
    if (x < width - 1) {
        float weight = lambda * weightx[offset];
        coeff += weight;
        sigma += weight * output[offset_c + nChannels];
    }

    // Update output value with solved equation
    if (coeff > 0)
        output[offset_c] = omega * (sigma / coeff) + (1 - omega) * output[offset_c];
}