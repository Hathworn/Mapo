#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Compute_weightx_weighty2_norm0_Kernel(float* weightx, float* weighty, const float* absIx, const float* absIy, int nPixels, float norm_for_smooth_term, float eps)
{
    // Utilize shared memory to reduce global memory access time
    extern __shared__ float sharedData[];
    float* sharedAbsIx = sharedData;
    float* sharedAbsIy = sharedData + blockDim.x;

    int tx = threadIdx.x;
    int x = blockIdx.x * blockDim.x + tx;
    
    if (x < nPixels) {
        // Load data into shared memory
        sharedAbsIx[tx] = absIx[x];
        sharedAbsIy[tx] = absIy[x];
        __syncthreads();  // Ensure all threads have loaded data

        // Perform computations using shared memory
        float absIx_val = sharedAbsIx[tx];
        float absIy_val = sharedAbsIy[tx];
        
        weightx[x] = 1.0f / (absIx_val * absIx_val + eps);
        weighty[x] = 1.0f / (absIy_val * absIy_val + eps);
    }
}