#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Solve_redblack1_Kernel(float* output, const float* input, int width, int height, int nChannels, int c, const float* weightdata, const float* weightx, const float* weighty, float lambda, float omega, bool redflag)
{
    // Calculate global thread indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Bounds check
    if (x >= width || y >= height || (y + x) % 2 == redflag) return;

    int offset = y * width + x;
    int slice = width * nChannels;
    int offset_c = offset * nChannels + c;
    float coeff = 0.0f, sigma = 0.0f, weight = 0.0f;

    // Use shared memory for weights to reduce global memory access
    __shared__ float shared_weightdata[512];
    __shared__ float shared_weightx[512];
    __shared__ float shared_weighty[512];

    // Load weights to shared memory
    int sharedIdx = threadIdx.y * blockDim.x + threadIdx.x;
    shared_weightdata[sharedIdx] = weightdata[offset];
    shared_weightx[sharedIdx] = weightx[offset];
    shared_weighty[sharedIdx] = weighty[offset];
    __syncthreads();

    // Accessing shared memory instead of global memory for weights
    if (y > 0)
    {
        weight = lambda * shared_weighty[sharedIdx - blockDim.x];
        coeff += weight;
        sigma += weight * output[offset_c - slice];
    }
    if (y < height - 1)
    {
        weight = lambda * shared_weighty[sharedIdx];
        coeff += weight;
        sigma += weight * output[offset_c + slice];
    }
    if (x > 0)
    {
        weight = lambda * shared_weightx[sharedIdx - 1];
        coeff += weight;
        sigma += weight * output[offset_c - nChannels];
    }
    if (x < width - 1)
    {
        weight = lambda * shared_weightx[sharedIdx];
        coeff += weight;
        sigma += weight * output[offset_c + nChannels];
    }

    coeff += shared_weightdata[sharedIdx];
    sigma += shared_weightdata[sharedIdx] * input[offset_c];
    
    // Ensuring coeff is positive if (coeff > 0) is satisfied
    if (coeff > 0.0f) 
        output[offset_c] = sigma / coeff * omega + output[offset_c] * (1.0f - omega);
}