#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Dx_Forward_Kernel(float* output, const float* input, const int width, const int height, const int nChannels)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Check if thread is within bounds
    if (x >= width || y >= height) return;

    int offset = y * width + x;
    int offsetChannels = offset * nChannels; // Pre-calculate for efficiency

    if (x == width - 1)
    {
        // Optimize assignment using memset
        for (int c = 0; c < nChannels; c++)
        {
            output[offsetChannels + c] = 0;
        }
    }
    else
    {
        for (int c = 0; c < nChannels; c++)
        {
            // Avoid recomputing offset
            output[offsetChannels + c] = input[(offset + 1) * nChannels + c] - input[offsetChannels + c];
        }
    }
}