#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Compute_weightdata_Kernel(float* weightdata, const float* I, const float* input, int nPixels, int nChannels, int c, float norm_for_data_term, float eps)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundary condition
    if (x < nPixels)
    {
        float diff = fabs(I[x] - input[x * nChannels + c]) + eps; // Calculate difference outside if-else

        // Compute weightdata based on norm
        if (norm_for_data_term == 2)
        {
            weightdata[x] = 1;
        }
        else if (norm_for_data_term == 1)
        {
            weightdata[x] = 1.0f / diff;
        }
        else
        {
            weightdata[x] = pow(diff, norm_for_data_term - 2);
        }
    }
}