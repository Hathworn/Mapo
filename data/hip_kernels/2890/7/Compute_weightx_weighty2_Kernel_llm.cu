#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Compute_weightx_weighty2_Kernel(float* weightx, float* weighty, const float* absIx, const float* absIy, int nPixels, float norm_for_smooth_term, float eps)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x >= nPixels) return;

    // Use conditional operator to simplify code
    float inv_eps = 1.0f / eps;
    float inv_norm = 1.0f / (2.0f - norm_for_smooth_term);

    if (norm_for_smooth_term == 2) {
        weightx[x] = 1.0f;
        weighty[x] = 1.0f;
    }
    else {
        float ix_val = absIx[x];
        float iy_val = absIy[x];
        if (norm_for_smooth_term == 1) {
            weightx[x] = 1.0f / (ix_val + eps);
            weighty[x] = 1.0f / (iy_val + eps);
        }
        else if (norm_for_smooth_term == 0) {
            weightx[x] = 1.0f / (ix_val * ix_val + eps);
            weighty[x] = 1.0f / (iy_val * iy_val + eps);
        }
        else {
            weightx[x] = pow(ix_val, inv_norm) * inv_eps;
            weighty[x] = pow(iy_val, inv_norm) * inv_eps;
        }
    }
}