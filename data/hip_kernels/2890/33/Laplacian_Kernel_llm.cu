#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Laplacian_Kernel(float* output, const float* input, const int width, const int height, const int nChannels)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= width || y >= height)
        return;
    
    int offset = y * width + x;

    // Use shared memory for better memory coalescing for input values
    extern __shared__ float sharedMem[];
    int sharedOffset = threadIdx.y * blockDim.x + threadIdx.x;

    // Load input data into shared memory
    for (int c = 0; c < nChannels; c++)
        sharedMem[sharedOffset * nChannels + c] = input[offset * nChannels + c];
    __syncthreads();

    float value;
    for (int c = 0; c < nChannels; c++)
    {
        value = 0.0f;
        
        // Using ternary operations for boundary checks
        value += (x < width - 1 ? sharedMem[(sharedOffset + 1) * nChannels + c] : 0.0f)
               + (x > 0 ? sharedMem[(sharedOffset - 1) * nChannels + c] : 0.0f)
               - 2 * sharedMem[sharedOffset * nChannels + c];

        value += (y < height - 1 ? sharedMem[(sharedOffset + blockDim.x) * nChannels + c] : 0.0f)
               + (y > 0 ? sharedMem[(sharedOffset - blockDim.x) * nChannels + c] : 0.0f)
               - 2 * sharedMem[sharedOffset * nChannels + c];

        output[offset * nChannels + c] = value;
    }
}