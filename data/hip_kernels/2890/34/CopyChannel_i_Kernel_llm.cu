#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CopyChannel_i_Kernel(float* output, const float* input, const int i, const int width, const int height, const int nChannels)
{
    // Calculate index for 1D traversal to maximize coalesced memory access
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Define total number of elements to process
    int totalElements = width * height;

    // Process elements by stride of blockDim.x * gridDim.x for efficient workload distribution
    for (int strideIndex = index; strideIndex < totalElements; strideIndex += blockDim.x * gridDim.x) {
        // Calculate 2D coordinates from the 1D index
        int x = strideIndex % width;
        int y = strideIndex / width;

        // Calculate offset for the current pixel
        int offset = y * width + x;

        // Copy the specific channel 'i' from input to output
        output[offset] = input[offset * nChannels + i];
    }
}