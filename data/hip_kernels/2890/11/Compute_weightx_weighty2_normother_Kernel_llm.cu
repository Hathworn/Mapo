#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Compute_weightx_weighty2_normother_Kernel(float* weightx, float* weighty, const float* absIx, const float* absIy, int nPixels, float norm_for_smooth_term, float eps)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Use conditional operator to reduce branching
    if (x < nPixels) {
        float pow_factor = 2.0f - norm_for_smooth_term;
        float absIx_val = absIx[x];
        float absIy_val = absIy[x];

        // Precomputing common expression
        float common_expr_x = powf(absIx_val, pow_factor) + eps;
        float common_expr_y = powf(absIy_val, pow_factor) + eps;

        weightx[x] = 1.0f / common_expr_x;
        weighty[x] = 1.0f / common_expr_y;
    }
}