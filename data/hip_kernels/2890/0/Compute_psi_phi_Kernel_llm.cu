#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Compute_psi_phi_Kernel(float* psi, float* phi, const float* gAbsIx, const float* gAbsIy, const float* gIx, const float* gIy, int nPixels, float norm_for_contrast_num, float norm_for_contrast_denom, float eps)
{
    // Calculate unique thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within the bounds
    if (x >= nPixels)
        return;

    float psi_num, psi_denom;
    float phi_num, phi_denom;

    // Use ternary operators for efficient calculations
    psi_num = (norm_for_contrast_num == 0) ? 1 :
              (norm_for_contrast_num == 1) ? gAbsIx[x] :
              (norm_for_contrast_num == 2) ? gAbsIx[x] * gAbsIx[x] :
              powf(gAbsIx[x], norm_for_contrast_num);

    phi_num = (norm_for_contrast_num == 0) ? 1 :
              (norm_for_contrast_num == 1) ? gAbsIy[x] :
              (norm_for_contrast_num == 2) ? gAbsIy[x] * gAbsIy[x] :
              powf(gAbsIy[x], norm_for_contrast_num);

    psi_denom = (norm_for_contrast_denom == 0) ? 1 :
                (norm_for_contrast_denom == 1) ? fabsf(gIx[x]) + eps :
                (norm_for_contrast_denom == 2) ? gIx[x] * gIx[x] + eps :
                powf(fabsf(gIx[x]), norm_for_contrast_denom) + eps;

    phi_denom = (norm_for_contrast_denom == 0) ? 1 :
                (norm_for_contrast_denom == 1) ? fabsf(gIy[x]) + eps :
                (norm_for_contrast_denom == 2) ? gIy[x] * gIy[x] + eps :
                powf(fabsf(gIy[x]), norm_for_contrast_denom) + eps;

    // Compute psi and phi
    psi[x] = psi_num / psi_denom;
    phi[x] = phi_num / phi_denom;
}