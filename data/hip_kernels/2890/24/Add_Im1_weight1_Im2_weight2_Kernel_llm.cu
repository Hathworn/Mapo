#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Add_Im1_weight1_Im2_weight2_Kernel(float* output, const float* Im1, const float weight1, const float* Im2, const float weight2, const int width, const int height, const int nChannels)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Use a single return statement with an early exit strategy
    if (x < width && y < height) { 
        int offset = (y * width + x) * nChannels; // Calculate offset once for better performance
        for (int c = 0; c < nChannels; c++) {
            output[offset + c] = Im1[offset + c] * weight1 + Im2[offset + c] * weight2; // Optimized memory access pattern
        }
    }
}