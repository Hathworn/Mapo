#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MedianFilterWithMask5x5_Kernel(float* output, const float* input, const int width, const int height, const int nChannels, const bool* keep_mask)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= width || y >= height)
        return;
    int offset = y * width + x;
    if (keep_mask[offset])
    {
        for (int c = 0; c < nChannels; c++)
            output[offset * nChannels + c] = input[offset * nChannels + c];
        return;
    }

    float vals[25];
    int count;
    for (int c = 0; c < nChannels; c++)
    {
        count = 0;
        int start_x = max(x - 2, 0);
        int end_x = min(x + 2, width - 1);
        int start_y = max(y - 2, 0);
        int end_y = min(y + 2, height - 1);

        for (int ii = start_y; ii <= end_y; ii++)
        {
            for (int jj = start_x; jj <= end_x; jj++)
            {
                int cur_offset = ii * width + jj;
                if (keep_mask[cur_offset])
                {
                    vals[count++] = input[cur_offset * nChannels + c];
                }
            }
        }

        if (count == 0)
        {
            output[offset * nChannels + c] = 0;
        }
        else
        {
            for (int i = 0; i < count - 1; i++) 
            {
                // Perform insertion sort
                int j = i + 1;
                float key = vals[j];
                while (j > 0 && vals[j - 1] > key) 
                {
                    vals[j] = vals[j - 1];
                    j--;
                }
                vals[j] = key;
            }

            output[offset * nChannels + c] = vals[count / 2];
        }
    }
}