#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MulWithScale_Kernel(float* in_out_put, const float scale, const int width, const int height, const int nChannels)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds once per thread
    if (x < width && y < height) {
        int offset = (y * width + x) * nChannels; // Calculate full offset including channels
        for (int c = 0; c < nChannels; c++) {
            in_out_put[offset + c] *= scale; // Multiply each channel element
        }
    }
}