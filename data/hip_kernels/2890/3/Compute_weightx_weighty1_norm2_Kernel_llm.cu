#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Compute_weightx_weighty1_norm2_Kernel(float* weightx, float* weighty, const float* psi, const float* phi, const float* absIx, const float* absIy, int nPixels, float norm_for_smooth_term, float eps)
{
    // Use shared memory for commonly accessed data to reduce global memory access latency
    extern __shared__ float shared_psi[];
    int tx = threadIdx.x;
    int x = blockIdx.x * blockDim.x + tx;

    if (x < nPixels)
    {
        // Load inputs to shared memory
        shared_psi[tx] = psi[x];
        __syncthreads();

        // Use shared memory for faster access
        weightx[x] = shared_psi[tx];
        weighty[x] = phi[x];  // Assumes phi doesn't benefit from shared memory based on access patterns
    }
}