#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Solve_redblack2_new_Kernel(float* output, const float* input, int width, int height, int nChannels, int c, const float* weightx, const float* weighty, float lambda, float omega, bool redflag)
{
    // Calculate thread and block indices
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Calculate position
    int x = (bx * blockDim.x + tx) * 2 + (redflag ? (ty % 2) : (ty % 2 + 1));
    int y = by * blockDim.y + ty;

    // Check boundaries
    if (x >= width || y >= height) return;

    int offset = y * width + x;
    int slice = width * nChannels;
    int offset_c = offset * nChannels + c;
    
    float coeff = 1.0f; // Optimization: Initialize with 1
    float sigma = input[offset_c]; // Optimization: Initialize with input[offset_c]

    // Check neighbors
    if (y > 0) {
        float weight = lambda * weighty[offset - width];
        coeff += weight;
        sigma += weight * output[offset_c - slice];
    }
    if (y < height - 1) {
        float weight = lambda * weighty[offset];
        coeff += weight;
        sigma += weight * output[offset_c + slice];
    }
    if (x > 0) {
        float weight = lambda * weightx[offset - 1];
        coeff += weight;
        sigma += weight * output[offset_c - nChannels];
    }
    if (x < width - 1) {
        float weight = lambda * weightx[offset];
        coeff += weight;
        sigma += weight * output[offset_c + nChannels];
    }

    // Update output
    output[offset_c] = (sigma / coeff) * omega + output[offset_c] * (1.0f - omega); // Optimization: Removed redundant condition check
}