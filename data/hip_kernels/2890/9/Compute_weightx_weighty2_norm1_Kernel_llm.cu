#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Compute_weightx_weighty2_norm1_Kernel(float* weightx, float* weighty, const float* absIx, const float* absIy, int nPixels, float norm_for_smooth_term, float eps)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < nPixels){
        // Use reciprocal to optimize division
        float inv_eps_x = 1.0f / (absIx[x] + eps);
        float inv_eps_y = 1.0f / (absIy[x] + eps);
        weightx[x] = inv_eps_x;
        weighty[x] = inv_eps_y;
    }
}