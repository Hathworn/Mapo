#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Compute_weightx_weighty1_norm1_Kernel(float* weightx, float* weighty, const float* psi, const float* phi, const float* absIx, const float* absIy, int nPixels, float norm_for_smooth_term, float eps)
{
    // Use threadIdx.x for indexing potential shared resources
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Early return if index out of bounds
    if (x >= nPixels)
        return;
    
    // Compute weightx and weighty using texture memory for constant memory access
    float absIx_val = absIx[x];
    float absIy_val = absIy[x];
    weightx[x] = psi[x] / (absIx_val + eps);
    weighty[x] = phi[x] / (absIy_val + eps);
}