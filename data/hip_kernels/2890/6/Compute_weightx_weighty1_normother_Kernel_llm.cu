#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Compute_weightx_weighty1_normother_Kernel(float* weightx, float* weighty, const float* psi, const float* phi, const float* absIx, const float* absIy, int nPixels, float norm_for_smooth_term, float eps)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure x is within bounds to eliminate warp divergence
    if (x < nPixels) {
        float absIx_val = absIx[x];  // Load global memory elements to register
        float absIy_val = absIy[x];  // Load global memory elements to register
        weightx[x] = psi[x] / (pow(absIx_val, 2.0f - norm_for_smooth_term) + eps);
        weighty[x] = phi[x] / (pow(absIy_val, 2.0f - norm_for_smooth_term) + eps);
    }
}