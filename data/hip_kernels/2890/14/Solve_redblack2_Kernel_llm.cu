#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Solve_redblack2_Kernel(float* __restrict__ output, const float* __restrict__ input, int width, int height, int nChannels, int c, const float* __restrict__ weightx, const float* __restrict__ weighty, float lambda, float omega, bool redflag)
{
    // Calculate indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary check
    if (x >= width || y >= height) return;

    // Check if the current cell is the correct color
    if ((y + x) % 2 == redflag) return;

    int offset = y * width + x;
    int slice = width * nChannels;
    int offset_c = offset * nChannels + c;

    // Initialize local variables
    float coeff = 1.0f;
    float sigma = input[offset_c];

    // Consolidate conditions and operations
    if (y > 0) {
        float weight = lambda * weighty[offset - width];
        coeff += weight;
        sigma += weight * output[offset_c - slice];
    }
    if (y < height - 1) {
        float weight = lambda * weighty[offset];
        coeff += weight;
        sigma += weight * output[offset_c + slice];
    }
    if (x > 0) {
        float weight = lambda * weightx[offset - 1];
        coeff += weight;
        sigma += weight * output[offset_c - nChannels];
    }
    if (x < width - 1) {
        float weight = lambda * weightx[offset];
        coeff += weight;
        sigma += weight * output[offset_c + nChannels];
    }

    // Update output using precomputed values
    output[offset_c] = omega * sigma / coeff + (1 - omega) * output[offset_c];
}