#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Compute_weightx_weighty1_norm0_Kernel(float* weightx, float* weighty, const float* psi, const float* phi, const float* absIx, const float* absIy, int nPixels, float norm_for_smooth_term, float eps)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread is within bounds
    if (x < nPixels) {
        float absIx_eps = absIx[x] * absIx[x] + eps; // Precompute commonly used expression
        float absIy_eps = absIy[x] * absIy[x] + eps; // Precompute commonly used expression

        weightx[x] = psi[x] / absIx_eps;
        weighty[x] = phi[x] / absIy_eps;
    }
}