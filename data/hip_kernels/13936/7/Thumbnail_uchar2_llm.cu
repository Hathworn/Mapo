#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {

texture<unsigned char, 2> uchar_tex;
texture<uchar2, 2>  uchar2_tex;
texture<unsigned short, 2> ushort_tex;
texture<ushort2, 2>  ushort2_tex;

}

__global__ void Thumbnail_uchar2(int *histogram, int src_width, int src_height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds before accessing texture memory for optimal branching
    if (x < src_width && y < src_height)
    {
        uchar2 pixel = tex2D(uchar2_tex, x, y);
        
        // Use shared memory to reduce global atomic operations
        __shared__ int local_hist[512];

        // Initialize the shared memory histogram
        if (threadIdx.x < 512) local_hist[threadIdx.x] = 0;
        __syncthreads();

        // Update local histogram
        atomicAdd(&local_hist[pixel.x], 1);
        atomicAdd(&local_hist[256 + pixel.y], 1);
        __syncthreads();

        // Write back to global memory
        if (threadIdx.x < 512)
            atomicAdd(&histogram[threadIdx.x], local_hist[threadIdx.x]);
    }
}