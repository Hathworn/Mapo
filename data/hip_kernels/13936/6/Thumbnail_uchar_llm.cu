#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {

texture<unsigned char, 2> uchar_tex;
texture<uchar2, 2>  uchar2_tex;
texture<unsigned short, 2> ushort_tex;
texture<ushort2, 2>  ushort2_tex;

}

__global__ void Thumbnail_uchar(int *histogram, int src_width, int src_height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Early exit if out of bounds
    if (y >= src_height || x >= src_width) return;

    unsigned char pixel = tex2D(uchar_tex, x, y);
    atomicAdd(&histogram[pixel], 1);
}