#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {

texture<unsigned char, 2> uchar_tex;
texture<uchar2, 2>  uchar2_tex;
texture<uchar4, 2>  uchar4_tex;
texture<unsigned short, 2> ushort_tex;
texture<ushort2, 2>  ushort2_tex;
texture<ushort4, 2>  ushort4_tex;


}

__global__ void Subsample_Bilinear_uchar4(uchar4 *dst, int dst_width, int dst_height, int dst_pitch, int src_width, int src_height) {
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width) {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;
        
        // Compute filter weights and derive bilinear weights
        float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
        float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);

        // Perform texture fetches
        uchar4 c0 = tex2D(uchar4_tex, xi - dx, yi - dy);
        uchar4 c1 = tex2D(uchar4_tex, xi + dx, yi - dy);
        uchar4 c2 = tex2D(uchar4_tex, xi - dx, yi + dy);
        uchar4 c3 = tex2D(uchar4_tex, xi + dx, yi + dy);

        // Compute weighted sum and type conversion
        int4 res;
        res.x = ((int)c0.x + (int)c1.x + (int)c2.x + (int)c3.x + 2) >> 2;
        res.y = ((int)c0.y + (int)c1.y + (int)c2.y + (int)c3.y + 2) >> 2;
        res.z = ((int)c0.z + (int)c1.z + (int)c2.z + (int)c3.z + 2) >> 2;
        res.w = ((int)c0.w + (int)c1.w + (int)c2.w + (int)c3.w + 2) >> 2;

        // Assign result to output
        dst[yo * dst_pitch + xo] = make_uchar4(
            (unsigned char)res.x, (unsigned char)res.y, (unsigned char)res.z, (unsigned char)res.w);
    }
}