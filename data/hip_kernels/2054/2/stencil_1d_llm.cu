#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1d(int *in, int *out){

    // Allocate shared memory for the computation with halo
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];

    int gindex = threadIdx.x + blockIdx.x * blockDim.x; // Global index
    int lindex = threadIdx.x + RADIUS; // Local index in shared memory with halo

    // Read input elements into shared memory
    // Critical to avoid bank conflicts is to access shared memory coalesced
    temp[lindex] = in[gindex]; // Center
    if (threadIdx.x < RADIUS) {
        temp[threadIdx.x] = in[gindex - RADIUS]; // Left halo
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE]; // Right halo
    }

    __syncthreads(); // Synchronize to make sure the data is available for all threads

    // Apply the stencil operation
    int result = 0;
    #pragma unroll
    for (int offset = -RADIUS; offset <= RADIUS; offset++) {
        result += temp[lindex + offset];
    }

    // Store the computed result to global memory
    out[gindex] = result;
}