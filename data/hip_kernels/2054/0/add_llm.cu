#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define N 512

__global__ void add(int *a, int *b, int *c) {
    // Use threadIdx.x for better granularity
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        c[index] = a[index] + b[index];  // Utilize more threads
    }
}