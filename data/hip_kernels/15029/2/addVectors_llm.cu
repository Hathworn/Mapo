#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addVectors(const int entries, const float *a, const float *b, float *ab) {
    // Calculate global index using blockDim
    int index = blockDim.x * blockIdx.x + threadIdx.x; 

    // Ensure the index is within bounds
    if (index < entries) {
        ab[index] = a[index] + b[index];
    }
}