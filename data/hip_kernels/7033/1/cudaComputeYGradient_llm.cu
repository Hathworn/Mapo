#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipEvent_t start, stop;

__global__ void cudaComputeYGradient(int* y_gradient, unsigned char* channel, int image_width, int image_height) {
    // Define the y_kernel locally in shared memory for faster access
    __shared__ int y_kernel[3][3];
    if (threadIdx.x < 3 && threadIdx.y < 3) { 
        int temp[3][3] = { { 1, 2, 1 }, { 0, 0, 0 }, { -1, -2, -1 } };
        y_kernel[threadIdx.x][threadIdx.y] = temp[threadIdx.x][threadIdx.y];
    }
    __syncthreads();
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check valid index bounds to avoid accessing out of bounds memory
    if (index >= image_width && index < (image_height - 1) * image_width && index % image_width != 0 && index % image_width != image_width - 1) {
        y_gradient[index] =
            y_kernel[0][0] * channel[index - 1] +
            y_kernel[1][0] * channel[index] +
            y_kernel[2][0] * channel[index + 1] +
            y_kernel[0][1] * channel[index + image_width - 1] +
            y_kernel[1][1] * channel[index + image_width] +
            y_kernel[2][1] * channel[index + image_width + 1] +
            y_kernel[0][2] * channel[index + 2 * image_width - 1] +
            y_kernel[1][2] * channel[index + 2 * image_width] +
            y_kernel[2][2] * channel[index + 2 * image_width + 1];
    }
}