#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaComputeYGradient(int* y_gradient, unsigned char* channel, int image_width, int image_height, int chunk_size_per_thread) {
    // Shared memory for tile loading
    __shared__ unsigned char tile[1024]; // Assume a block size of 1024 threads, adjust accordingly

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int block_start = index * chunk_size_per_thread;
    int block_end = (index + 1) * chunk_size_per_thread - 1;

    // Load data into shared memory
    int local_index = threadIdx.x;
    if (block_start + local_index < image_width * image_height) {
        tile[local_index] = channel[block_start + local_index];
    }
    __syncthreads();

    for (int i = block_start; i < block_end; i++) {
        if (i + 2 * image_width + 1 < image_width * image_height) {
            if (i == 0 && blockIdx.x == 0 && threadIdx.x == 0) {
                continue;
            } else {
                int value = 
                    tile[local_index - 1] + 2 * tile[local_index] + tile[local_index + 1] +
                    channel[i + image_width - 1] + 2 * channel[i + image_width] + channel[i + image_width + 1] +
                    channel[i + 2 * image_width - 1] + 2 * channel[i + 2 * image_width] + channel[i + 2 * image_width + 1];
                y_gradient[i] = value - (
                    tile[local_index - 1] + tile[local_index] + tile[local_index + 1] + 
                    channel[i + image_width - 1] + channel[i + image_width] + channel[i + image_width + 1] +
                    channel[i + 2 * image_width - 1] + channel[i + 2 * image_width] + channel[i + 2 * image_width + 1]);
            }
        }
    }
    return;
}