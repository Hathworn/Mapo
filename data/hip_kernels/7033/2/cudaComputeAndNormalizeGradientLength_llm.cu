#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipEvent_t start, stop;

__global__ void cudaComputeAndNormalizeGradientLength(unsigned char *channel_values, int* x_gradient, int* y_gradient) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use local variables to store reused values
    int x_grad = x_gradient[index];
    int y_grad = y_gradient[index];
    
    // Calculate gradient length using faster approximation
    int gradient_length = rsqrtf(float(x_grad * x_grad + y_grad * y_grad)) * 255.0f;

    // Clamp the gradient length to 255
    gradient_length = min(gradient_length, 255);

    // Save the computed gradient length
    channel_values[index] = static_cast<unsigned char>(gradient_length);
}