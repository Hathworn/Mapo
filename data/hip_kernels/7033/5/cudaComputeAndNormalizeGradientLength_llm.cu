#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaComputeAndNormalizeGradientLength(unsigned char *channel_values, int* x_gradient, int* y_gradient, int chunk_size_per_thread) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Utilize shared memory for potential performance gains
    extern __shared__ int shared_mem[];

    for (int i = index * chunk_size_per_thread; i < (index + 1) * chunk_size_per_thread; i++) {
        // Calculate gradient length using shared memory for x and y gradients
        int x_grad = x_gradient[i];
        int y_grad = y_gradient[i];
        int gradient_length = int(sqrtf(float(x_grad * x_grad + y_grad * y_grad)));

        // Clamp the gradient length to 255 using min function
        channel_values[i] = min(gradient_length, 255);
    }
    return;
}
```
