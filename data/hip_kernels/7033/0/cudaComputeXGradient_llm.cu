#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipEvent_t start, stop;

__global__ void cudaComputeXGradient(int* x_gradient, unsigned char* channel, int image_width, int image_height) {
    // Preload kernel coefficients into registers for faster access
    const int x_kernel[3][3] = { { 1, 0, -1 }, { 2, 0, -2 }, { 1, 0, -1 } };
    
    // Compute the global index and row/col position
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int row = index / image_width;
    int col = index % image_width;
    
    // Ensure index is within bounds of valid computation area
    if (row >= 1 && row < image_height - 1 && col >= 1 && col < image_width - 1) {
        // Compute gradient using shared memory for better memory access (assuming block size allows this)
        int gradient = 0;
        for (int i = -1; i <= 1; ++i) {
            for (int j = -1; j <= 1; ++j) {
                int neighbor_index = index + i * image_width + j;
                gradient += x_kernel[i + 1][j + 1] * channel[neighbor_index];
            }
        }
        x_gradient[index] = gradient;
    }
}