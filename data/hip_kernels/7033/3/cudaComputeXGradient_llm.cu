#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaComputeXGradient(int* x_gradient, unsigned char* channel, int image_width, int image_height, int chunk_size_per_thread) {
    __shared__ int x_kernel[3][3]; // Use shared memory for kernel
    if (threadIdx.x == 0) {
        x_kernel[0][0] = 1; x_kernel[0][1] = 0; x_kernel[0][2] = -1;
        x_kernel[1][0] = 2; x_kernel[1][1] = 0; x_kernel[1][2] = -2;
        x_kernel[2][0] = 1; x_kernel[2][1] = 0; x_kernel[2][2] = -1;
    }
    __syncthreads(); // Synchronize threads to ensure kernel is initialized

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int start_idx = index * chunk_size_per_thread;
    int end_idx = min(start_idx + chunk_size_per_thread, image_width * image_height - 1);

    for (int i = start_idx; i < end_idx; i++) {
        if (i >= image_width && i < image_width * (image_height - 1) - 1) { // Ensure the pixel is not at the border
            x_gradient[i] =
                x_kernel[0][0] * channel[i - 1] +
                x_kernel[1][0] * channel[i] +
                x_kernel[2][0] * channel[i + 1] +
                x_kernel[0][1] * channel[i + image_width - 1] +
                x_kernel[1][1] * channel[i + image_width] +
                x_kernel[2][1] * channel[i + image_width + 1] +
                x_kernel[0][2] * channel[i + 2 * image_width - 1] +
                x_kernel[1][2] * channel[i + 2 * image_width] +
                x_kernel[2][2] * channel[i + 2 * image_width + 1];
        }
    }
}