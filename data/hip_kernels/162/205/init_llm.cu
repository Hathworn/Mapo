#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(int *vector, int N, int val)
{
    // Calculate global thread index with stride
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // Loop to cover all elements with fewer threads
    for (; i < N; i += blockDim.x * gridDim.x) {
        vector[i] = val;
    }
}