#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void entrySearch_max_int_kernel(int *g_iarr, int *g_maxarr, int size)
{
    // Create shared memory
    extern __shared__ int sarr_int[];

    // Calculate thread and block indices
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    // Load elements into shared memory
    int local_max = INT_MIN;
    if (i < size) {
        local_max = g_iarr[i];
        if (i + blockDim.x < size) {
            local_max = max(local_max, g_iarr[i + blockDim.x]);
        }
    }
    sarr_int[tid] = local_max;
    __syncthreads();

    // Parallel reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sarr_int[tid] = max(sarr_int[tid], sarr_int[tid + s]);
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        g_maxarr[blockIdx.x] = sarr_int[0];
    }
}