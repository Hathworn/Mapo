#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bin_start(int *binStart, int *binEnd, int *partBin, int nparts)
{
    // This kernel function was adapted from NVIDIA CUDA 5.5 Examples
    // This software contains source code provided by NVIDIA Corporation

    extern __shared__ int sharedBin[];  // blockSize + 1
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (index < nparts) {
        int bin = partBin[index];
        sharedBin[threadIdx.x + 1] = bin;

        // Load the neighbor particle bin for the first thread in the block
        if (threadIdx.x == 0) {
            sharedBin[0] = (index > 0) ? partBin[index - 1] : -1; // Use -1 if index is 0
        }
    }
    __syncthreads();

    if (index < nparts) {
        int bin = partBin[index];
        
        // Handle first particle in a bin
        if (index == 0 || bin != sharedBin[threadIdx.x]) {
            binStart[bin] = index;
            if (index > 0) {
                binEnd[sharedBin[threadIdx.x]] = index;
            }
        }

        // Special case for the last particle
        if (index == nparts - 1) {
            binEnd[bin] = index + 1;
        }
    }
}