#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to initialize hiprand states
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Compute global index
    hiprand_init(seed, idx, 0, &state[idx]); // Use global index for state initialization
}