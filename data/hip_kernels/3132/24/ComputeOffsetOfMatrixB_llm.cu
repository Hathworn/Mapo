#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Compute global thread index
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use grid stride loop to handle all elements when number of threads is less than N
    for (int32_t i = idx; i < N * gridDim.x; i += blockDim.x * gridDim.x) {
        int32_t row = i / N;
        int32_t col = i % N;
        // Efficiently assign the negative of row_sum value to the correct position in output
        output[row * N + col] = -row_sum[row];
    }
}