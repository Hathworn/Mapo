#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PadMatrixInLeadingDimensionKernel(const int8_t* src, int8_t* dst, int col_src, int col_dst) {
    int tid = threadIdx.x;
    int block_offset_src = blockIdx.x * col_src;
    int block_offset_dst = blockIdx.x * col_dst;
    
    // Loop unrolling for efficiency, assuming col_src is a multiple of 4
    #pragma unroll
    for (int32_t i = tid; i < col_src; i += blockDim.x) {
        dst[block_offset_dst + i] = src[block_offset_src + i];
    }
}