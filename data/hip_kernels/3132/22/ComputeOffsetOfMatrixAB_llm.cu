#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixAB(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, int32_t K_A_B, int32_t N) {
    // Calculate a unique index for each thread
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x; 
    
    // Use a stride loop to efficiently cover the array
    for (int32_t i = idx; i < N; i += blockDim.x * gridDim.x) {
        output[blockIdx.x * N + i] = K_A_B - row_sum[blockIdx.x] - col_sum[i];
    }
}