#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (idx < N) { // Ensure thread index is within bounds
        output[blockIdx.x * N + idx] = -col_sum[idx];
    }
}