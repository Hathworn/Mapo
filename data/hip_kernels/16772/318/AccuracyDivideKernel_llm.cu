#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Execute only for thread index 0 to handle division once
    if (idx == 0) {
        *accuracy /= N;
    }
}