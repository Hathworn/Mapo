#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use atomic operation for safe division in parallel execution
    extern __shared__ float temp[];
    int tid = threadIdx.x;
    
    if (tid == 0) {
        temp[0] = *accuracy;
    }
    
    __syncthreads();
    
    // Divide shared value by N
    if (tid == 0) {
        temp[0] /= N;
    }
    
    __syncthreads();
    
    // Store result back to global memory
    if (tid == 0) {
        *accuracy = temp[0];
    }
}