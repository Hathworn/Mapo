#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use atomic operation to ensure thread safety when updating *accuracy
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    float current_accuracy = *accuracy;
    atomicExch(accuracy, current_accuracy / N);
}