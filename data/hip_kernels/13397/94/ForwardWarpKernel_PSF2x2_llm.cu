#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate the position for current pixel
    float cx = u[flow_row_offset + j] * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + static_cast<float>(i) + 1.0f;

    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);
    
    float value = src[image_row_offset + j];
    
    // Atomic operations for updating the destination pixel and normalization factor
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        float weight = dx * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Update for bottom left corner
    tx -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        float weight = (1.0f - dx) * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Update for upper left corner
    ty -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        float weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Update for upper right corner
    tx += 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        float weight = dx * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}