#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Utilize row-major memory coalescing
    if (i < h && j < w)
    {
        image[i * w + j] = value;
    }
}