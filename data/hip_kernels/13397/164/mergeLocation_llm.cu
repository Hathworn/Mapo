#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate global thread index
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if within bounds and perform calculations
    if (ptidx < npoints)
    {
        // Load short2 element into registers
        short2 loc = loc_[ptidx];

        // Use FMA to directly calculate and store scaled values
        x[ptidx] = __fmul_rd(loc.x, scale);
        y[ptidx] = __fmul_rd(loc.y, scale);
    }
}