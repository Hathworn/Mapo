#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_index = i * flow_stride + j;
    int image_index = i * image_stride + j;

    float u_ = u[flow_index];
    float v_ = v[flow_index];

    // Calculate target coordinates
    float cx = u_ * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v_ * time_scale + static_cast<float>(i) + 1.0f;

    int tx = __float2int_rn(cx); // Target x-coordinate
    int ty = __float2int_rn(cy); // Target y-coordinate

    float value = src[image_index]; // Source pixel value

    // Use atomic add only if target is within bounds
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        _atomicAdd(dst + ty * image_stride + tx, value);
    }
}