#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t sortWithCuda(int *a, size_t size, float* time);

typedef long long int64;
typedef unsigned long long uint64;

__global__ void swapOnKernel(int *a, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x * 2;
    int cacheFirst, cacheSecond, cacheThird;

    for (int j = 0; j < size/2 + 1; j++) {

        if(i+1 < size) {
            cacheFirst = a[i];
            cacheSecond = a[i+1];

            // Swap if cacheFirst is greater than cacheSecond
            if(cacheFirst > cacheSecond) {
                int temp = cacheFirst;
                a[i] = cacheSecond;
                cacheSecond = a[i+1] = temp;
            }
        }

        if(i+2 < size) {
            cacheThird = a[i+2];
            
            // Swap if cacheSecond is greater than cacheThird
            if(cacheSecond > cacheThird) {
                int temp = cacheSecond;
                a[i+1] = cacheThird;
                a[i+2] = temp;
            }
        }

        __syncthreads();  // Synchronize threads after processing each segment
    }
}