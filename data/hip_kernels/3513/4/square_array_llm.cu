#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float* input, float* output, int n) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (idx < n) {
        // Perform squaring operation
        output[idx] = input[idx] * input[idx];
    }
}