#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* __restrict__ d_input_cdf, float* __restrict__ d_output_cdf, int n)
{
    const float normalization_constant = 1.f / d_input_cdf[n - 1];
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_index_1d < n)
    {
        // Load input value once
        unsigned int input_value = d_input_cdf[global_index_1d];

        // Compute output and store directly
        d_output_cdf[global_index_1d] = input_value * normalization_constant;
    }
}