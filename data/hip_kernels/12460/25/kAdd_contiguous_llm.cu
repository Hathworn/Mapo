#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Use shared memory to optimize memory access and reduce global memory access latency
    __shared__ float a_shared[256];
    __shared__ float b_shared[256];

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        // Load data into shared memory for the current block
        a_shared[threadIdx.x] = a[i];
        b_shared[threadIdx.x] = b[i];
        __syncthreads(); // Ensure all threads have loaded data to shared memory

        // Compute result using shared memory
        dest[i] = a_shared[threadIdx.x] + b_shared[threadIdx.x];
        __syncthreads(); // Ensure all computations are done before next iteration
    }
}