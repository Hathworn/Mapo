#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Simplified and optimized kernel function for readability and efficiency
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Check if there is a single thread to perform the division safely
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N; 
    }
}