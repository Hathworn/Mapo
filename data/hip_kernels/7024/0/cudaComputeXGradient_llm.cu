#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipEvent_t start, stop;

__global__ void cudaComputeXGradient(int* x_gradient, unsigned char* channel, int image_width, int image_height) {
    __shared__ int x_kernel[3][3];
    if (threadIdx.x < 3 && threadIdx.y < 3) {
        int local_x_kernel[3][3] = { 
            { 1, 0, -1 }, 
            { 2, 0, -2 }, 
            { 1, 0, -1 } 
        };
        x_kernel[threadIdx.x][threadIdx.y] = local_x_kernel[threadIdx.x][threadIdx.y];
    }
    __syncthreads(); // Ensure all threads have loaded the kernel

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx <= 0 || idx >= image_width * image_height - 1) {
        return;
    }

    int gx = 0;
    for (int dy = -1; dy <= 1; ++dy) {
        for (int dx = -1; dx <= 1; ++dx) {
            int cx = idx % image_width + dx;
            int cy = idx / image_width + dy;
            if (cx >= 0 && cx < image_width && cy >= 0 && cy < image_height) {
                int c_idx = cy * image_width + cx;
                gx += x_kernel[dy + 1][dx + 1] * channel[c_idx];
            }
        }
    }
    x_gradient[idx] = gx;
}