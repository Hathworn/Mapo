#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaComputeYGradient(int* y_gradient, unsigned char* channel, int image_width, int image_height, int chunk_size_per_thread) {
    // Use shared memory for y_kernel to optimize memory access
    __shared__ int y_kernel[3][3];
    if (threadIdx.x == 0) {
        y_kernel[0][0] = 1; y_kernel[0][1] = 2; y_kernel[0][2] = 1;
        y_kernel[1][0] = 0; y_kernel[1][1] = 0; y_kernel[1][2] = 0;
        y_kernel[2][0] = -1; y_kernel[2][1] = -2; y_kernel[2][2] = -1;
    }
    __syncthreads();

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int start = index * chunk_size_per_thread;
    int end = min(start + chunk_size_per_thread, image_width * image_height - 1);

    for (int i = start; i < end; i++) {
        if (i + 2 * image_width + 1 < image_width * image_height) {
            if (!(i == 0 && blockIdx.x == 0 && threadIdx.x == 0)) { // Correct thread check
                y_gradient[i] =
                    y_kernel[0][0] * channel[i - 1] +
                    y_kernel[1][0] * channel[i] +
                    y_kernel[2][0] * channel[i + 1] +
                    y_kernel[0][1] * channel[i + image_width - 1] +
                    y_kernel[1][1] * channel[i + image_width] +
                    y_kernel[2][1] * channel[i + image_width + 1] +
                    y_kernel[0][2] * channel[i + 2 * image_width - 1] +
                    y_kernel[1][2] * channel[i + 2 * image_width] +
                    y_kernel[2][2] * channel[i + 2 * image_width + 1];
            }
        }
    }
}