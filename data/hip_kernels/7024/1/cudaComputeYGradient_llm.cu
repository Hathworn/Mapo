#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

hipEvent_t start, stop;

__global__ void cudaComputeYGradient(int* y_gradient, unsigned char* channel, int image_width, int image_height) {
    // Shared memory to avoid global memory access within a block
    __shared__ unsigned char shared_channel[3 * 1024];

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int sharedMemIdx = threadIdx.x;

    if (index >= image_width * image_height) return; // Boundary check

    // Load data into shared memory
    if (sharedMemIdx < 3 * blockDim.x) {
        shared_channel[sharedMemIdx] = channel[index - 1 + sharedMemIdx];
    }
    __syncthreads();

    // Compute gradient only for valid threads
    if (index > 0 && index < image_width * image_height - 1) {
        int y_kernel[3][3] = { { 1, 2, 1 }, { 0, 0, 0 }, { -1, -2, -1 } };

        y_gradient[index] = y_kernel[0][0] * shared_channel[sharedMemIdx] +
                            y_kernel[1][0] * shared_channel[sharedMemIdx + 1] +
                            y_kernel[2][0] * shared_channel[sharedMemIdx + 2] +
                            y_kernel[0][1] * shared_channel[sharedMemIdx + blockDim.x] +
                            y_kernel[1][1] * shared_channel[sharedMemIdx + blockDim.x + 1] +
                            y_kernel[2][1] * shared_channel[sharedMemIdx + blockDim.x + 2] +
                            y_kernel[0][2] * shared_channel[sharedMemIdx + 2 * blockDim.x] +
                            y_kernel[1][2] * shared_channel[sharedMemIdx + 2 * blockDim.x + 1] +
                            y_kernel[2][2] * shared_channel[sharedMemIdx + 2 * blockDim.x + 2];
    }
}