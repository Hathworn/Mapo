#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaComputeAndNormalizeGradientLength(unsigned char *channel_values, int* x_gradient, int* y_gradient, int chunk_size_per_thread) {
    // Calculate the index for thread and check bounds
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int start = index * chunk_size_per_thread;
    int end = start + chunk_size_per_thread;

    // Unroll loop for better performance
    #pragma unroll
    for (int i = start; i < end; i++) {
        // Use built-in square root function
        int gradient_length = min(255, int(hypot(float(x_gradient[i]), float(y_gradient[i]))));
        channel_values[i] = gradient_length;
    }
}