#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipEvent_t start, stop;

__global__ void cudaComputeAndNormalizeGradientLength(unsigned char *channel_values, int* x_gradient, int* y_gradient) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory for faster access within block (assuming input size is known and fits)
    __shared__ int shared_x[256];
    __shared__ int shared_y[256];
    
    // Load data into shared memory
    shared_x[threadIdx.x] = x_gradient[index];
    shared_y[threadIdx.x] = y_gradient[index];
    __syncthreads();  // Ensure all threads have loaded their data

    // Compute gradient length
    int x_grad = shared_x[threadIdx.x];
    int y_grad = shared_y[threadIdx.x];
    int gradient_length = __fsqrt_rn(float(x_grad * x_grad + y_grad * y_grad));

    // Normalize and store result
    channel_values[index] = min(gradient_length, 255);
}