#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaComputeXGradient(int* x_gradient, unsigned char* channel, int image_width, int image_height, int chunk_size_per_thread) {
    __shared__ unsigned char shared_channel[1024]; // Using shared memory for faster access
    int x_kernel[3][3] = { { 1, 0, -1 }, { 2, 0, -2 }, { 1, 0, -1 } };

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int start = index * chunk_size_per_thread;
    int end = min(image_width * image_height, (index + 1) * chunk_size_per_thread);

    for (int i = start; i < end; i++) {
        if (i + 2 * image_width + 1 < image_width * image_height) {
            // Load data into shared memory
            shared_channel[threadIdx.x] = channel[i];
            __syncthreads(); // Make sure all threads have loaded their data

            if (i > 0 && i % image_width != 0 && i % image_width != image_width - 1) {
                x_gradient[i] =
                    x_kernel[0][0] * shared_channel[threadIdx.x - 1] +
                    x_kernel[1][0] * shared_channel[threadIdx.x] +
                    x_kernel[2][0] * shared_channel[threadIdx.x + 1] +
                    x_kernel[0][1] * shared_channel[threadIdx.x + blockDim.x - 1] +
                    x_kernel[1][1] * shared_channel[threadIdx.x + blockDim.x] +
                    x_kernel[2][1] * shared_channel[threadIdx.x + blockDim.x + 1] +
                    x_kernel[0][2] * shared_channel[threadIdx.x + 2 * blockDim.x - 1] +
                    x_kernel[1][2] * shared_channel[threadIdx.x + 2 * blockDim.x] +
                    x_kernel[2][2] * shared_channel[threadIdx.x + 2 * blockDim.x + 1];
            }
        }
    }
}