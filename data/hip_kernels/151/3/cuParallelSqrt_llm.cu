#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// If the code is used in Matlab, set MATLAB_CODE to 1. Otherwise, set MATLAB_CODE to 0.
#define MATLAB_CODE 0

// Constants used by the program
#define MAX_PITCH_VALUE_IN_BYTES       262144
#define MAX_TEXTURE_WIDTH_IN_BYTES     65536
#define MAX_TEXTURE_HEIGHT_IN_BYTES    32768
#define MAX_PART_OF_FREE_MEMORY_USED   0.9
#define BLOCK_DIM                      16

// Texture containing the reference points (if it is possible)
texture<float, 2, hipReadModeElementType> texA;

__global__ void cuParallelSqrt(float *dist, int width) {
    // Calculate global index
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (xIndex < width) {
        // Use the rsqrt function for a faster approximation of the square root
        dist[xIndex] = rsqrtf(dist[xIndex]) * dist[xIndex];
    }
}