#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuAddRNorm(float *dist, int width, int pitch, int height, float *vec) {
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    // Load shared data cooperatively
    __shared__ float shared_vec[16];
    if (threadIdx.x == 0 && yIndex < height) {
        shared_vec[threadIdx.y] = vec[yIndex];
    }
    __syncthreads();

    // Perform computation if indices are within bounds
    if (xIndex < width && yIndex < height) {
        dist[yIndex * pitch + xIndex] += shared_vec[threadIdx.y];
    }
}