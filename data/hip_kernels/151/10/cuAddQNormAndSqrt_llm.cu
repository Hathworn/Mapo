#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuAddQNormAndSqrt(float *vec1, float *vec2, int width) {
    // Use shared memory for faster access (if possible with data size)
    extern __shared__ float sharedVec1[];

    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (xIndex < width) {
        // Load data into shared memory for faster access
        sharedVec1[threadIdx.x] = vec1[xIndex];
        
        // Use intrinsic function for sqrt (if supported)
        float val = __sqrtf(sharedVec1[threadIdx.x] + vec2[xIndex]);

        // Store result back to global memory
        vec1[xIndex] = val;
    }
}