#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_distances(float * ref, int ref_width, int ref_pitch, float * query, int query_width, int query_pitch, int height, float * dist) {

    // Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
    __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Block index
    int blockX = blockIdx.x;
    int blockY = blockIdx.y;

    // Initializarion of the SSD for the current thread
    float ssd = 0.f;

    // Loop parameters
    int begin_A = BLOCK_DIM * blockY * ref_pitch;
    int begin_B = BLOCK_DIM * blockX * query_pitch;

    // Conditions
    int cond0 = (tx + BLOCK_DIM * blockY < ref_width); // used to write in shared memory
    int cond1 = (tx + BLOCK_DIM * blockX < query_width); // used to write in shared memory & to computations and to write in output array
    int cond2 = (ty + BLOCK_DIM * blockY < ref_width); // used to computations and to write in output matrix

    // Loop over all the sub-matrices of A and B required to compute the block sub-matrix
    for (int a = begin_A, b = begin_B, end_A = begin_A + (height - 1) * ref_pitch; a <= end_A; a += BLOCK_DIM * ref_pitch, b += BLOCK_DIM * query_pitch) {

        // Load the matrices from device memory to shared memory; each thread loads one element of each matrix
        if (a / ref_pitch + ty < height) {
            shared_A[ty][tx] = (cond0) ? ref[a + ref_pitch * ty + tx] : 0;
            shared_B[ty][tx] = (cond1) ? query[b + query_pitch * ty + tx] : 0;
        }
        else {
            shared_A[ty][tx] = 0;
            shared_B[ty][tx] = 0;
        }

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Compute the difference between the two matrices; each thread computes one element of the block sub-matrix
        if (cond2 && cond1) {
            #pragma unroll  // Unroll the loop for better performance
            for (int k = 0; k < BLOCK_DIM; ++k) {
                float tmp = shared_A[k][ty] - shared_B[k][tx];
                ssd += tmp*tmp;
            }
        }

        // Synchronize to make sure that the preceding computation is done before loading two new sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory; each thread writes one element
    if (cond2 && cond1) {
        dist[(BLOCK_DIM * blockY + ty) * query_pitch + BLOCK_DIM * blockX + tx] = ssd;
    }
}