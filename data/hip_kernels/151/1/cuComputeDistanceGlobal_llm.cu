#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// If the code is used in Matlab, set MATLAB_CODE to 1. Otherwise, set MATLAB_CODE to 0.
#define MATLAB_CODE 0

// Includes
#if MATLAB_CODE == 1
#else
#endif

// Constants used by the program
#define MAX_PITCH_VALUE_IN_BYTES       262144
#define MAX_TEXTURE_WIDTH_IN_BYTES     65536
#define MAX_TEXTURE_HEIGHT_IN_BYTES    32768
#define MAX_PART_OF_FREE_MEMORY_USED   0.9
#define BLOCK_DIM                      16

// Texture containing the reference points (if it is possible)
texture<float, 2, hipReadModeElementType> texA;

//-----------------------------------------------------------------------------------------------//
//                                            KERNELS                                            //
//-----------------------------------------------------------------------------------------------//

/**
* Computes the distance between two matrix A (reference points) and
* B (query points) containing respectively wA and wB points.
*
* @param A     pointer on the matrix A
* @param wA    width of the matrix A = number of points in A
* @param pA    pitch of matrix A given in number of columns
* @param B     pointer on the matrix B
* @param wB    width of the matrix B = number of points in B
* @param pB    pitch of matrix B given in number of columns
* @param dim   dimension of points = height of matrices A and B
* @param AB    pointer on the matrix containing the wA*wB distances computed
*/

// Optimized kernel: remove unnecessary shared variable declarations
__global__ void cuComputeDistanceGlobal(float* A, int wA, int pA, float* B, int wB, int pB, int dim, float* AB) {
  __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
  __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  float ssd = 0;
  int begin_A = BLOCK_DIM * blockIdx.y;
  int begin_B = BLOCK_DIM * blockIdx.x;
  int step_A  = BLOCK_DIM * pA;
  int step_B  = BLOCK_DIM * pB;
  int end_A   = begin_A + (dim - 1) * pA;

  bool cond0 = (begin_A + tx < wA); 
  bool cond1 = (begin_B + tx < wB);
  bool cond2 = (begin_A + ty < wA);

  for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {
    // Optimized with ternary operators for readability and performance
    shared_A[ty][tx] = ((a/pA + ty < dim) && cond0) ? A[a + pA * ty + tx] : 0;
    shared_B[ty][tx] = ((a/pA + ty < dim) && cond1) ? B[b + pB * ty + tx] : 0;

    __syncthreads();

    if (cond2 && cond1) {
      #pragma unroll // Performance optimization: unroll loop
      for (int k = 0; k < BLOCK_DIM; ++k) {
        float tmp = shared_A[k][ty] - shared_B[k][tx];
        ssd += tmp * tmp;
      }
    }
    __syncthreads();
  }

  if (cond2 && cond1) {
    AB[(begin_A + ty) * pB + begin_B + tx] = ssd;
  }
}