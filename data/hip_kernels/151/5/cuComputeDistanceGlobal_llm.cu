#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuComputeDistanceGlobal(float* A, int wA, int pA, float* B, int wB, int pB, int dim, float* AB) {
    __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int begin_A = BLOCK_DIM * blockIdx.y;
    int begin_B = BLOCK_DIM * blockIdx.x;
    int step_A = BLOCK_DIM * pA;
    int step_B = BLOCK_DIM * pB;
    int end_A = begin_A + (dim - 1) * pA;

    bool cond0 = (begin_A + tx < wA);
    bool cond1 = (begin_B + tx < wB);
    bool cond2 = (begin_A + ty < wA);

    float ssd = 0.0f;

    // Loop over all sub-matrices of A and B required to compute the block sub-matrix
    for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {
        // Load matrices from device memory to shared memory
        if (a / pA + ty < dim) {
            shared_A[ty][tx] = cond0 ? A[a + pA * ty + tx] : 0;
            shared_B[ty][tx] = cond1 ? B[b + pB * ty + tx] : 0;
        } else {
            shared_A[ty][tx] = 0;
            shared_B[ty][tx] = 0;
        }

        __syncthreads();  // Ensure matrices are loaded

        // Compute difference between matrices
        if (cond2 && cond1) {
            #pragma unroll  // Unroll loop for better performance
            for (int k = 0; k < BLOCK_DIM; ++k) {
                float tmp = shared_A[k][ty] - shared_B[k][tx];
                ssd += tmp * tmp;
            }
        }

        __syncthreads();  // Ensure preceding computation is done
    }

    // Write the block sub-matrix to device memory
    if (cond2 && cond1) {
        AB[(begin_A + ty) * pB + begin_B + tx] = ssd;
    }
}