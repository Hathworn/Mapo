#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuComputeNorm(float *mat, int width, int pitch, int height, float *norm) {
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if xIndex is within bounds
    if (xIndex < width) {
        float sum = 0.0f;

        // Use shared memory for improved data locality
        extern __shared__ float shared[];

        // Load matrix values into shared memory in chunks
        for (int i = threadIdx.y; i < height; i += blockDim.y) {
            shared[threadIdx.y] = mat[i * pitch + xIndex];
            __syncthreads();
            sum += shared[threadIdx.y] * shared[threadIdx.y];
            __syncthreads();
        }

        // Write the sum to norm array
        norm[xIndex] = sum;
    }
}