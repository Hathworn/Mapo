#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuAddRNorm(float *dist, int width, int pitch, int height, float *vec) {
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int xIndex = blockIdx.x * blockDim.x + tx;
    unsigned int yIndex = blockIdx.y * blockDim.y + ty;

    // Use dynamic shared memory and only load vector values once per block
    extern __shared__ float shared_vec[];
    if (tx == 0 && yIndex < height)
        shared_vec[ty] = vec[yIndex];
    __syncthreads();

    // Ensure all threads have valid indices
    if (xIndex < width && yIndex < height) {
        dist[yIndex * pitch + xIndex] += shared_vec[ty];
    }
}