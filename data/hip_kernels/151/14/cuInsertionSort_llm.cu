#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuInsertionSort(float *dist, int dist_pitch, int *ind, int ind_pitch, int width, int height, int k) {
    // Assign thread index
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if the thread is within bounds
    if (xIndex < width) {
        // Pointer setup
        float *p_dist = dist + xIndex;
        int *p_ind = ind + xIndex;
        
        // Initialize first element
        float max_dist = p_dist[0];
        p_ind[0] = 1;
        
        // Part 1: Sort the first k elements
        for (int l = 1; l < k; l++) {
            int curr_row = l * dist_pitch;
            float curr_dist = p_dist[curr_row];
            
            // Optimized insertion sort for the first k elements
            if (curr_dist < max_dist) {
                int i = l - 1;
                while (i >= 0 && p_dist[i * dist_pitch] > curr_dist) {
                    i--;
                }
                i++;
                for (int j = l; j > i; j--) {
                    p_dist[j * dist_pitch] = p_dist[(j - 1) * dist_pitch];
                    p_ind[j * ind_pitch] = p_ind[(j - 1) * ind_pitch];
                }
                p_dist[i * dist_pitch] = curr_dist;
                p_ind[i * ind_pitch] = l + 1;
                max_dist = p_dist[(k - 1) * dist_pitch];
            } else {
                p_ind[l * ind_pitch] = l + 1;
            }
        }
        
        // Part 2: Insert remaining elements
        for (int l = k; l < height; l++) {
            int curr_row = l * dist_pitch;
            float curr_dist = p_dist[curr_row];
            
            // Insert only if the current distance is smaller
            if (curr_dist < max_dist) {
                int i = k - 1;
                while (i >= 0 && p_dist[i * dist_pitch] > curr_dist) {
                    i--;
                }
                i++;
                for (int j = k - 1; j > i; j--) {
                    p_dist[j * dist_pitch] = p_dist[(j - 1) * dist_pitch];
                    p_ind[j * ind_pitch] = p_ind[(j - 1) * ind_pitch];
                }
                p_dist[i * dist_pitch] = curr_dist;
                p_ind[i * ind_pitch] = l + 1;
                max_dist = p_dist[(k - 1) * dist_pitch];
            }
        }
    }
}