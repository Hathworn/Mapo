#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MATLAB_CODE 0
#define MAX_PITCH_VALUE_IN_BYTES       262144
#define MAX_TEXTURE_WIDTH_IN_BYTES     65536
#define MAX_TEXTURE_HEIGHT_IN_BYTES    32768
#define MAX_PART_OF_FREE_MEMORY_USED   0.9
#define BLOCK_DIM                      16

texture<float, 2, hipReadModeElementType> texA;

__global__ void cuComputeDistanceTexture(int wA, float * B, int wB, int pB, int dim, float* AB) {
    // Calculate the x and y indices
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIndex < wB && yIndex < wA) {
        float ssd = 0;
        // Unroll the loop for improved performance
        for (int i = 0; i < dim; i += 4) {
            float tmp1 = tex2D(texA, (float)yIndex, (float)i) - B[i * pB + xIndex];
            ssd += tmp1 * tmp1;
            if (i + 1 < dim) {
                float tmp2 = tex2D(texA, (float)yIndex, (float)(i + 1)) - B[(i + 1) * pB + xIndex];
                ssd += tmp2 * tmp2;
            }
            if (i + 2 < dim) {
                float tmp3 = tex2D(texA, (float)yIndex, (float)(i + 2)) - B[(i + 2) * pB + xIndex];
                ssd += tmp3 * tmp3;
            }
            if (i + 3 < dim) {
                float tmp4 = tex2D(texA, (float)yIndex, (float)(i + 3)) - B[(i + 3) * pB + xIndex];
                ssd += tmp4 * tmp4;
            }
        }
        // Write the computed distance to the output matrix
        AB[yIndex * pB + xIndex] = ssd;
    }
}