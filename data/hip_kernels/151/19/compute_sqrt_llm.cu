#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_sqrt(float *dist, int width, int pitch, int k) {
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread indices are within bounds
    if (xIndex < width && yIndex < k) {
        unsigned int index = yIndex * pitch + xIndex;

        // Load from global memory, compute sqrt and store back
        float val = dist[index];
        dist[index] = sqrtf(val); // Use sqrtf for single precision
    }
}