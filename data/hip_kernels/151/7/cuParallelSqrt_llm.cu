#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuParallelSqrt(float *dist, int width, int pitch, int k) {
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check if within bounds
    if (xIndex < width && yIndex < k) {
        // Use registers for repeated access
        unsigned int index = yIndex * pitch + xIndex;
        float value = dist[index];
        dist[index] = sqrt(value);
    }
}