#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuComputeNorm(float *mat, int width, int pitch, int height, float *norm){
    // Cache block index and thread index
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (xIndex < width){
        float sum = 0;
        // Use shared memory for faster access
        __shared__ float shared_val[1024]; // Edit size according to block size

        // Load values into shared memory
        for (int i = threadIdx.y; i < height; i += blockDim.y) {
            shared_val[threadIdx.y] = mat[i * pitch + xIndex];
        }
        __syncthreads();

        // Compute sum of squares
        for (int i = threadIdx.y; i < height; i += blockDim.y) {
            float val = shared_val[threadIdx.y];
            sum += val * val;
        }

        // Store result
        if (threadIdx.y == 0) {
            norm[xIndex] = sum;
        }
    }
}