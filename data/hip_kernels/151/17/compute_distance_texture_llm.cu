#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_distance_texture(hipTextureObject_t ref, int ref_width, float* query, int query_width, int query_pitch, int height, float* dist) {
    // Calculate global thread indices
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIndex < query_width && yIndex < ref_width) {
        float ssd = 0.f;
        
        // Unroll the loop for better performance
        int i = 0;
        #pragma unroll
        for (; i + 3 < height; i += 4) {
            float tmp0 = tex2D<float>(ref, (float)yIndex, (float)i) - query[i * query_pitch + xIndex];
            float tmp1 = tex2D<float>(ref, (float)yIndex, (float)(i + 1)) - query[(i + 1) * query_pitch + xIndex];
            float tmp2 = tex2D<float>(ref, (float)yIndex, (float)(i + 2)) - query[(i + 2) * query_pitch + xIndex];
            float tmp3 = tex2D<float>(ref, (float)yIndex, (float)(i + 3)) - query[(i + 3) * query_pitch + xIndex];
            ssd += tmp0 * tmp0 + tmp1 * tmp1 + tmp2 * tmp2 + tmp3 * tmp3;
        }
        
        // Handle remaining iterations
        for (; i < height; i++) {
            float tmp = tex2D<float>(ref, (float)yIndex, (float)i) - query[i * query_pitch + xIndex];
            ssd += tmp * tmp;
        }
        
        // Store the computed sum of squared differences
        dist[yIndex * query_pitch + xIndex] = ssd;
    }
}