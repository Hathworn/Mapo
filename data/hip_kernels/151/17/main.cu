#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "compute_distance_texture.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    hipTextureObject_t ref = 1;
int ref_width = XSIZE;
float *query = NULL;
hipMalloc(&query, XSIZE*YSIZE);
int query_width = XSIZE;
int query_pitch = 2;
int height = YSIZE;
float *dist = NULL;
hipMalloc(&dist, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
compute_distance_texture<<<gridBlock, threadBlock>>>(ref,ref_width,query,query_width,query_pitch,height,dist);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
compute_distance_texture<<<gridBlock, threadBlock>>>(ref,ref_width,query,query_width,query_pitch,height,dist);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
compute_distance_texture<<<gridBlock, threadBlock>>>(ref,ref_width,query,query_width,query_pitch,height,dist);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}