#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_squared_norm(float * array, int width, int pitch, int height, float * norm){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (xIndex < width) {
        float sum = 0.f;
        // Use shared memory to reduce global memory access
        extern __shared__ float shared_array[];
        
        for (int i = threadIdx.y; i < height; i += blockDim.y) {
            shared_array[threadIdx.y] = array[i * pitch + xIndex];
            __syncthreads();  // Synchronize to ensure all threads have loaded their data
            
            float val = shared_array[threadIdx.y];
            sum += val * val;
            __syncthreads();  // Synchronize for safe shared memory reuse
        }
        
        // Reduce within a warp (assume blockDim.x is warpSize)
        for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
            sum += __shfl_down_sync(0xffffffff, sum, offset);
        }
        
        // Write the result for each warp's leader
        if ((threadIdx.y & (warpSize - 1)) == 0) {
            atomicAdd(&norm[xIndex], sum);
        }
    }
}