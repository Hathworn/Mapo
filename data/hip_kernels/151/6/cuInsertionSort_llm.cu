#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuInsertionSort(float *dist, int dist_pitch, int *ind, int ind_pitch, int width, int height, int k) {
    // Variables
    int l, i, j;
    float *p_dist;
    int *p_ind;
    float curr_dist, max_dist;
    int curr_row, max_row;
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (xIndex < width) {
        // Pointer shift, initialization, and max value
        p_dist = dist + xIndex;
        p_ind = ind + xIndex;
        max_dist = p_dist[0];
        p_ind[0] = 1;

        // Part 1: sort first k elements
        for (l = 1; l < k; l++) {
            curr_row = l * dist_pitch;
            curr_dist = p_dist[curr_row];
            if (curr_dist < max_dist) {
                // Use binary search for finding position `i` for insertion
                int low = 0, high = l;
                while (low < high) {
                    int mid = (low + high) / 2;
                    if (p_dist[mid * dist_pitch] > curr_dist) {
                        high = mid;
                    } else {
                        low = mid + 1;
                    }
                }
                i = low;
                
                for (j = l; j > i; j--) {
                    p_dist[j * dist_pitch] = p_dist[(j - 1) * dist_pitch];
                    p_ind[j * ind_pitch] = p_ind[(j - 1) * ind_pitch];
                }
                p_dist[i * dist_pitch] = curr_dist;
                p_ind[i * ind_pitch] = l + 1;
            } else {
                p_ind[l * ind_pitch] = l + 1;
            }
            max_dist = p_dist[curr_row];
        }

        // Part 2: insert element in the first k lines
        max_row = (k - 1) * dist_pitch;
        for (l = k; l < height; l++) {
            curr_dist = p_dist[l * dist_pitch];
            if (curr_dist < max_dist) {
                // Use binary search for finding position `i` for insertion
                int low = 0, high = k;
                while (low < high) {
                    int mid = (low + high) / 2;
                    if (p_dist[mid * dist_pitch] > curr_dist) {
                        high = mid;
                    } else {
                        low = mid + 1;
                    }
                }
                i = low;
                
                for (j = k - 1; j > i; j--) {
                    p_dist[j * dist_pitch] = p_dist[(j - 1) * dist_pitch];
                    p_ind[j * ind_pitch] = p_ind[(j - 1) * ind_pitch];
                }
                p_dist[i * dist_pitch] = curr_dist;
                p_ind[i * ind_pitch] = l + 1;
                max_dist = p_dist[max_row];
            }
        }
    }
}