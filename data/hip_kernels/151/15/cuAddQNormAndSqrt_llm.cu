#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory to optimize data access and reduce global memory latency
__global__ void cuAddQNormAndSqrt(float *dist, int width, int pitch, float *q, int k) {
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIndex < width && yIndex < k) {
        // Load q[xIndex] into shared memory
        __shared__ float shared_q[256];  // Assumes blockDim.x <= 256, adjust as needed
        if (threadIdx.y == 0) {
            shared_q[threadIdx.x] = q[xIndex];
        }
        __syncthreads();

        // Use shared memory to perform the computation
        float q_value = shared_q[threadIdx.x];
        dist[yIndex * pitch + xIndex] = sqrt(dist[yIndex * pitch + xIndex] + q_value);
    }
}