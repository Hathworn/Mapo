#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuInsertionSort(float *dist, int width, int pitch, int height, int k) {
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (xIndex < width) {
        // Use registers for frequently accessed data
        float *p = dist + xIndex;
        float max_value = *p;

        // Part 1: Sort the first k elements
        for (int l = pitch; l < k * pitch; l += pitch) {
            float v = *(p + l);
            if (v < max_value) {
                int i = 0;

                // Improved loop with minimal iteration
                for (; i < l && *(p + i) <= v; i += pitch);

                // Use backward movement for better cache usage
                for (int j = l; j > i; j -= pitch)
                    *(p + j) = *(p + j - pitch);
                *(p + i) = v;
            }
            max_value = fmaxf(max_value, *(p + l)); // Use intrinsic function for max
        }

        // Part 2: Insert elements in the sorted k elements
        for (int l = k * pitch; l < height * pitch; l += pitch) {
            float v = *(p + l);
            if (v < max_value) {
                int i = 0;
                for (; i < k * pitch && *(p + i) <= v; i += pitch);

                for (int j = (k - 1) * pitch; j > i; j -= pitch)
                    *(p + j) = *(p + j - pitch);
                *(p + i) = v;
                max_value = *(p + (k - 1) * pitch);
            }
        }
    }
}