#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_query_points_norm_and_sqrt(float * array, int width, int pitch, int k, float * norm) {
    // Calculate indices
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (xIndex < width && yIndex < k) {
        // Use shared memory for norm to reduce memory accesses
        __shared__ float sharedNorm[blockDim.x];
        if (threadIdx.y == 0) {
            sharedNorm[threadIdx.x] = norm[xIndex];
        }
        __syncthreads();

        // Perform computation
        int index = yIndex * pitch + xIndex;
        array[index] = sqrt(array[index] + sharedNorm[threadIdx.x]);
    }
}