#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define FMC2W 16
#define FMC2H 4

#define M7W   32
#define M7H   32
#define M7R    4
#define NRX    2
#define NDIM 128

#define FMC_GH  512
#define FMC_BW   32
#define FMC_BH   32
#define FMC_BD   16
#define FMC_TW    1
#define FMC_TH    4
#define FMC_NW   (FMC_BW/FMC_TW)
#define FMC_NH   (FMC_BH/FMC_TH)
#define FMC_NT   (FMC_NW*FMC_NH)

__device__ volatile int lock = 0;

template <int size>
__device__ void InvertMatrix(float elem[size][size], float res[size][size]) {
    int indx[size];
    float b[size];
    float vv[size];
    for (int i = 0; i < size; i++)
        indx[i] = 0;

    int imax = 0;
    float d = 1.0;
    
    for (int i = 0; i < size; i++) {
        float big = 0.0;
        for (int j = 0; j < size; j++) {
            float temp = fabs(elem[i][j]);
            if (temp > big)
                big = temp;
        }
        if (big > 0.0)
            vv[i] = 1.0 / big;
        else
            vv[i] = 1e16;
    }

    for (int j = 0; j < size; j++) {
        for (int i = 0; i < j; i++) {
            float sum = elem[i][j];
            for (int k = 0; k < i; k++)
                sum -= elem[i][k] * elem[k][j];
            elem[i][j] = sum;
        }
        float big = 0.0;
        for (int i = j; i < size; i++) {
            float sum = elem[i][j];
            for (int k = 0; k < j; k++)
                sum -= elem[i][k] * elem[k][j];
            elem[i][j] = sum;
            float dum = vv[i] * fabs(sum);
            if (dum >= big) {
                big = dum;
                imax = i;
            }
        }
        if (j != imax) {
            for (int k = 0; k < size; k++) {
                float dum = elem[imax][k];
                elem[imax][k] = elem[j][k];
                elem[j][k] = dum;
            }
            d = -d;
            vv[imax] = vv[j];
        }
        indx[j] = imax;
        if (elem[j][j] == 0.0)
            elem[j][j] = 1e-16;
        if (j != (size - 1)) {
            float dum = 1.0 / elem[j][j];
            for (int i = j + 1; i < size; i++)
                elem[i][j] *= dum;
        }
    }

    for (int j = 0; j < size; j++) {
        for (int k = 0; k < size; k++)
            b[k] = 0.0;
        b[j] = 1.0;
        int ii = -1;
        for (int i = 0; i < size; i++) {
            int ip = indx[i];
            float sum = b[ip];
            b[ip] = b[i];
            if (ii != -1) {
                for (int j = ii; j < i; j++)
                    sum -= elem[i][j] * b[j];
            } else if (sum != 0.0) {
                ii = i;
            }
            b[i] = sum;
        }
        for (int i = size - 1; i >= 0; i--) {
            float sum = b[i];
            for (int j = i + 1; j < size; j++)
                sum -= elem[i][j] * b[j];
            b[i] = sum / elem[i][i];
        }
        for (int i = 0; i < size; i++)
            res[i][j] = b[i];
    }
}

__global__ void ComputeHomographies(float *coord, int *randPts, float *homo, int numPts) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int numLoops = blockDim.x * gridDim.x;

    float a[8][8] = {0}, ia[8][8], b[8] = {0};

    // Loop unrolling and prefetching points
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        int pt = randPts[i * numLoops + idx];
        float x1 = coord[pt];
        float y1 = coord[pt + numPts];
        float x2 = coord[pt + 2 * numPts];
        float y2 = coord[pt + 3 * numPts];
        
        // Setup matrix 'a' and vector 'b'
        float *row1 = a[2 * i];
        row1[0] = x1; row1[1] = y1; row1[2] = 1.0;
        row1[6] = -x2 * x1; row1[7] = -x2 * y1;
        float *row2 = a[2 * i + 1];
        row2[3] = x1; row2[4] = y1; row2[5] = 1.0;
        row2[6] = -y2 * x1; row2[7] = -y2 * y1;
        
        b[2 * i] = x2; b[2 * i + 1] = y2;
    }
    
    InvertMatrix<8>(a, ia);

    // Compute homography
    #pragma unroll
    for (int j = 0; j < 8; j++) {
        float sum = 0.0f;
        #pragma unroll
        for (int i = 0; i < 8; i++)
            sum += ia[j][i] * b[i];
        homo[j * numLoops + idx] = sum;
    }
}
```
