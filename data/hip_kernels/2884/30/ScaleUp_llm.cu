#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ScaleUp(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
    // Use variables directly to minimize repeated calculations
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int x = blockIdx.x * SCALEUP_W + 2 * tx;
    const int y = blockIdx.y * SCALEUP_H + 2 * ty;

    // Check bounds once for entire work under this thread
    if (x < 2 * width && y < 2 * height) {
        const int xl = blockIdx.x * (SCALEUP_W / 2) + tx;
        const int yu = blockIdx.y * (SCALEUP_H / 2) + ty;
        const int xr = min(xl + 1, width - 1);
        const int yd = min(yu + 1, height - 1);

        // Load data once to reduce global memory accesses
        const float vul = d_Data[yu * pitch + xl];
        const float vur = d_Data[yu * pitch + xr];
        const float vdl = d_Data[yd * pitch + xl];
        const float vdr = d_Data[yd * pitch + xr];

        // Calculate and store results directly
        const int index1 = y * newpitch + x;
        const int index2 = (y + 1) * newpitch + x;

        d_Result[index1 + 0] = vul;
        d_Result[index1 + 1] = 0.50f * (vul + vur);
        d_Result[index2 + 0] = 0.50f * (vul + vdl);
        d_Result[index2 + 1] = 0.25f * (vul + vur + vdl + vdr);
    }
}