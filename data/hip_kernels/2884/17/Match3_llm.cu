#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Match3(float *d_pts1, float *d_pts2, float *d_score, int *d_index)
{
    __shared__ float buffer1[M2W*(NDIM + 1)];
    __shared__ float buffer2[M2H*NDIM];
    __shared__ float scores[M2W*M2H];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int idx = tx + M2W*ty;
    int bp1 = M2W*blockIdx.x;

    // Load d_pts1 into shared memory buffer1
    if (ty < M2H) {
        #pragma unroll
        for (int j = ty; j < M2W; j += M2H) {
            #pragma unroll
            for (int d = tx; d < NDIM; d += M2W) {
                buffer1[j*(NDIM + 1) + d] = d_pts1[(bp1 + j)*NDIM + d];
            }
        }
    }
    __syncthreads();

    float max_score = 0.0f;
    int index = -1;

    // Loop over d_pts2 in blocks of M2H
    for (int bp2 = 0; bp2 < NPTS; bp2 += M2H) {
        // Load d_pts2 into shared memory buffer2
        #pragma unroll
        for (int d = tx; d < NDIM; d += M2W) {
            buffer2[ty*NDIM + d] = d_pts2[(bp2 + ty)*NDIM + d];
        }
        __syncthreads();

        float score = 0.0f;
        
        // Calculate score
        #pragma unroll
        for (int d = 0; d < NDIM; d++) {
            score += buffer1[tx*(NDIM + 1) + d] * buffer2[ty*NDIM + d];
        }
        scores[idx] = score;
        __syncthreads();

        // Reduce scores within the warp
        if (ty == 0) {
            float local_max = max_score;
            int local_index = index;

            #pragma unroll
            for (int i = 0; i < M2H; i++) {
                float temp_score = scores[i*M2W + tx];
                if (temp_score > local_max) {
                    local_max = temp_score;
                    local_index = bp2 + i;
                }
            }

            max_score = local_max;
            index = local_index;
        }
        __syncthreads();
    }

    // Write back the result
    if (ty == 0) {
        d_score[bp1 + tx] = max_score;
        d_index[bp1 + tx] = index;
    }
}