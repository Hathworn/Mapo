#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Match2(float *d_pts1, float *d_pts2, float *d_score, int *d_index)
{
    __shared__ float buffer1[M2W*NDIM];
    __shared__ float buffer2[M2H*NDIM];
    __shared__ float scores[M2W*M2H];
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bp1 = M2W*blockIdx.x;

    // Optimize loading of buffer1: Ensure coalesced access
    for (int d=ty; d<NDIM; d+=M2H)
        buffer1[tx*NDIM + d] = d_pts1[(bp1 + tx)*NDIM + d];
    __syncthreads();

    float max_score = 0.0f;
    int index = -1;

    // Optimize buffer2 loading and score calculation
    for (int bp2=0; bp2<NPTS; bp2+=M2H) {
        for (int d=ty; d<NDIM; d+=M2H)
            buffer2[tx*NDIM + d] = d_pts2[(bp2 + tx)*NDIM + d];
        __syncthreads();

        float score = 0.0f;
        for (int d=0; d<NDIM; d++)
            score += buffer1[tx*NDIM + d] * buffer2[ty*NDIM + d]; 

        atomicMax(&scores[tx], score);  // Reduce scores using atomicMax
        __syncthreads();

        if (tx == 0 && scores[ty] > max_score) {
            max_score = scores[ty];
            index = bp2 + ty;
        }
        __syncthreads();
    }

    if (tx == 0) {
        d_score[bp1 + ty] = max_score;
        d_index[bp1 + ty] = index;
    }
}