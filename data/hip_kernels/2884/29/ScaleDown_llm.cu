#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ScaleDown(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
    __shared__ float inrow[SCALEDOWN_W+4];
    __shared__ float brow[5*(SCALEDOWN_W/2)];
    __shared__ int yRead[SCALEDOWN_H+4];
    __shared__ int yWrite[SCALEDOWN_H+4];
    #define dx2 (SCALEDOWN_W/2)

    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * SCALEDOWN_W;
    const int yStart = blockIdx.y * SCALEDOWN_H;
    const int xWrite = xStart / 2 + tx;
    float k0 = d_ScaleDownKernel[0];
    float k1 = d_ScaleDownKernel[1];
    float k2 = d_ScaleDownKernel[2];

    if (tx < SCALEDOWN_H + 4) {
        int y = yStart + tx - 2;
        y = (y < 0 ? 0 : y);
        y = (y >= height ? height - 1 : y);
        yRead[tx] = y * pitch;
        yWrite[tx] = (yStart + tx - 4) / 2 * newpitch;
    }
    __syncthreads();

    int xRead = xStart + tx - 2;
    xRead = (xRead < 0 ? 0 : xRead);
    xRead = (xRead >= width ? width - 1 : xRead);

    int maxtx = min(dx2, width / 2 - xStart / 2);

    for (int dy = 0; dy < SCALEDOWN_H + 4; dy += 5) {
        // Load input data and compute partial reduction
        if (dy < (SCALEDOWN_H+4)) {
            inrow[tx] = d_Data[yRead[dy] + xRead];
            __syncthreads();
            if (tx < maxtx) {
                brow[(dy/5)*dx2 + tx] = k0*(inrow[2*tx] + inrow[2*tx+4]) + k1*(inrow[2*tx+1] + inrow[2*tx+3]) + k2*inrow[2*tx+2];
                
                // Calculate output applying the finalized kernel
                if (dy >= 4 && ((dy&1) == 0))
                    d_Result[yWrite[dy] + xWrite] = k2*brow[((dy/5)-1)*dx2 + tx] + k0*(brow[((dy/5)-2)*dx2 + tx] + brow[(dy/5)*dx2 + tx]) + k1*(brow[((dy/5)-1)*dx2 + tx] + brow[((dy/5)+1)*dx2 + tx]);
            }
            __syncthreads();
        }
    }
}