#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Match8(float *d_pts1, float *d_pts2, float *d_score, int *d_index)
{
    // Using shared memory for better load efficiency
    __shared__ float4 buffer1[M7W*NDIM/4];
    __shared__ float4 buffer2[M7H*NDIM/4];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int bp1 = M7W*blockIdx.x;
    
    // Load data into shared memory efficiently
    for (int d = tx; d < NDIM/4; d += M7W)
        for (int j = ty; j < M7W; j += M7H/M7R)
            buffer1[j*NDIM/4 + d] = ((float4*)d_pts1)[(bp1 + j)*(NDIM/4) + d];
    
    #define NRX 2
    float max_score[NRX];
    int index[NRX];
    
    for (int i = 0; i < NRX; i++) {
        max_score[i] = 0.0f;
        index[i] = -1;
    }
    
    int idx = ty*M7W + tx;
    int ix = idx % (M7W/NRX);
    int iy = idx / (M7W/NRX);
    
    for (int bp2 = 0; bp2 < NPTS; bp2 += M7H) {
        // Load data into shared memory efficiently
        for (int d = tx; d < NDIM/4; d += M7W)
            for (int j = ty; j < M7H; j += M7H/M7R)
                buffer2[j*NDIM/4 + d] = ((float4*)d_pts2)[(bp2 + j)*(NDIM/4) + d];
        
        __syncthreads();
        
        if (idx < M7W*M7H/M7R/NRX) {
            float score[M7R][NRX] = {0}; // Initialize score array directly
            
            for (int d = 0; d < NDIM/4; d++) {
                float4 v1[NRX];
                #pragma unroll // Unroll for better ILP
                for (int i = 0; i < NRX; i++)
                    v1[i] = buffer1[((M7W/NRX)*i + ix)*NDIM/4 + d];
                
                for (int dy = 0; dy < M7R; dy++) {
                    float4 v2 = buffer2[(M7R*iy + dy)*(NDIM/4) + d];
                    for (int i = 0; i < NRX; i++) {
                        // Vectorized accumulation
                        score[dy][i] += dot(v1[i], v2);
                    }
                }
            }
            // Reduce results and update max_score and index
            for (int dy = 0; dy < M7R; dy++) {
                for (int i = 0; i < NRX; i++) {
                    if (score[dy][i] > max_score[i]) {
                        max_score[i] = score[dy][i];
                        index[i] = bp2 + M7R*iy + dy;
                    }
                }
            }
        }
        __syncthreads();
    }
    
    float *scores = (float*)buffer1;
    int *indices = (int*)&scores[M7W*M7H/M7R];
    
    if (idx < M7W*M7H/M7R/NRX) {
        for (int i = 0; i < NRX; i++) {
            scores[iy*M7W + (M7W/NRX)*i + ix] = max_score[i];
            indices[iy*M7W + (M7W/NRX)*i + ix] = index[i];
        }
    }
    __syncthreads();
    
    if (ty == 0) {
        float max_score = scores[tx];
        int index = indices[tx];
        for (int y = 0; y < M7H/M7R; y++)
            if (scores[y*M7W + tx] > max_score) {
                max_score = scores[y*M7W + tx];
                index = indices[y*M7W + tx];
            }
        d_score[bp1 + tx] = max_score;
        d_index[bp1 + tx] = index;
    }
}