#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Match7(float *d_pts1, float *d_pts2, float *d_score, int *d_index)
{
    __shared__ float4 buffer1[M7W*NDIM/4]; 
    __shared__ float4 buffer2[M7H*NDIM/4];
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bp1 = M7W*blockIdx.x;

    // Optimize inner loop with unrolling
    for (int d=tx; d<NDIM/4; d+=M7W) {
        int d_off = d + tx;

        if (d_off < NDIM/4) {
            for (int j = ty; j < M7W; j += M7H/M7R) { 
                int idx = (bp1 + j)*(NDIM/4) + d;
                buffer1[j*NDIM/4 + d_off] = ((float4*)d_pts1)[idx];
            }
        }
    }

    float max_score = 0.0f;
    int index = -1;
    for (int bp2=0; bp2<NPTS; bp2+=M7H) {
        for (int d=tx; d<NDIM/4; d+=M7W) {
            int d_off = d;

            if (d_off < NDIM/4) {
                for (int j = ty; j < M7H; j += M7H/M7R) {
                    int idx = (bp2 + j)*(NDIM/4) + d;
                    buffer2[j*NDIM/4 + d_off] = ((float4*)d_pts2)[idx];
                }
            }
        }
        __syncthreads();

        float score[M7R] = {0}; // Initialize score array with zeros
        for (int d=0; d<NDIM/4; d++) {
            float4 v1 = buffer1[tx*NDIM/4 + (d + tx)%(NDIM/4)];
            for (int dy=0; dy<M7R; dy++) {
                float4 v2 = buffer2[(M7R*ty + dy)*(NDIM/4) + d];
                // Vectorized dot product
                score[dy] += dot(v1, v2);
            }
        }
        for (int dy=0; dy<M7R; dy++) {
            if (score[dy] > max_score) {
                max_score = score[dy];
                index = bp2 + M7R*ty + dy;
            }
        }
        __syncthreads();
    }

    float *scores = (float*)buffer1;
    int *indices = (int*)&scores[M7W*M7H/M7R];
    scores[ty*M7W + tx] = max_score;
    indices[ty*M7W + tx] = index;
    __syncthreads();

    if (ty == 0) {
        max_score = scores[tx];
        index = indices[tx];
        for (int y=0; y<M7H/M7R; y++)
            if (scores[y*M7W + tx] > max_score) {
                max_score = scores[y*M7W + tx];
                index = indices[y*M7W + tx];
            }
        d_score[bp1 + tx] = max_score;
        d_index[bp1 + tx] = index;
    }
}