#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Check for out-of-bounds access
    if(index < gridDim.x * blockDim.x) {
        c[index] = a[index] + b[index];
    }
}