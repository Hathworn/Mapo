#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    // Load values into shared memory for faster access
    __shared__ int shared_a, shared_b;

    // Only the first thread in the block loads data to shared memory
    if (threadIdx.x == 0) { 
        shared_a = *a;
        shared_b = *b;
    }
    
    __syncthreads();  // Ensure all threads see loaded values

    // First thread computes the result
    if (threadIdx.x == 0) {
        *c = shared_a + shared_b;
    }
}