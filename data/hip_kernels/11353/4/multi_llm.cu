#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multi(int *a, int *b, int *c, int n) {
    // Use shared memory to reduce global memory access latency
    __shared__ int shared_a[TILE_SIZE][TILE_SIZE];
    __shared__ int shared_b[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int suma = 0;

    for (int t = 0; t < (n + TILE_SIZE - 1) / TILE_SIZE; ++t) {
        // Load data into shared memory
        if (row < n && t * TILE_SIZE + threadIdx.x < n) {
            shared_a[threadIdx.y][threadIdx.x] = a[row * n + t * TILE_SIZE + threadIdx.x];
        } else {
            shared_a[threadIdx.y][threadIdx.x] = 0;
        }

        if (col < n && t * TILE_SIZE + threadIdx.y < n) {
            shared_b[threadIdx.y][threadIdx.x] = b[(t * TILE_SIZE + threadIdx.y) * n + col];
        } else {
            shared_b[threadIdx.y][threadIdx.x] = 0;
        }

        __syncthreads();

        // Perform computation using shared memory
        for (int i = 0; i < TILE_SIZE; ++i) {
            suma += shared_a[threadIdx.y][i] * shared_b[i][threadIdx.x];
        }

        __syncthreads();
    }

    // Write result back to global memory
    if (row < n && col < n) {
        c[row * n + col] = suma;
    }
}