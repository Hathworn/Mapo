#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Kernel2(bool *g_graph_mask, bool *g_updating_graph_mask, bool *g_graph_visited, bool *g_over, int no_of_nodes) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Early exit if tid is out of bounds
    if (tid >= no_of_nodes) return;
    
    // Check the condition to minimize divergent branches
    if (g_updating_graph_mask[tid]) {
        g_graph_mask[tid] = true;
        g_graph_visited[tid] = true;
        *g_over = true;
        g_updating_graph_mask[tid] = false;
    }
}