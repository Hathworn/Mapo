#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for better access speed if thread block size is small
    extern __shared__ short2 shmem[];
    if (ptidx < npoints)
    {
        shmem[threadIdx.x] = loc_[ptidx];
        __syncthreads();

        // Efficiently access shared memory
        short2 loc = shmem[threadIdx.x];
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}