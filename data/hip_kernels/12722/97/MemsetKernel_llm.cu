#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global thread index for linear memory access
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure the global index falls within the dimensions of the image
    if (idx < w && idy < h) {
        const int pos = idy * w + idx;
        image[pos] = value;
    }
}