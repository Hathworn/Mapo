#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;

    // Return early if out of bounds
    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate bottom-left corner of target pixel
    float cx = u[flow_row_offset + j] * time_scale + j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + i + 1.0f;

    // Separate integer and fractional parts
    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);

    // Target pixel integer coordinates
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);

    float value = src[image_row_offset + j];
    float weight;

    // Check boundaries and fill pixels with atomicAdd
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        weight = dx * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Bottom-left corner pixel
    tx -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        weight = (1.0f - dx) * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Upper-left corner pixel
    ty -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Upper-right corner pixel
    tx += 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        weight = dx * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}