#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global thread index
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Bound check to ensure threads operate within image dimensions
    if (i < h && j < w) {
        const int pos = i * s + j;

        // Avoid branch by using a multiplier
        float scale = normalization_factor[pos];        
        float invScale = (scale == 0.0f) ? 1.0f : __fdividef(1.0f, scale); // Use fast division

        // Apply normalization
        image[pos] *= invScale;
    }
}