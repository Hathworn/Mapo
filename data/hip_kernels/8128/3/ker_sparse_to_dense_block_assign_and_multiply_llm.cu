#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_sparse_to_dense_block_assign_and_multiply(int n, const unsigned *idx, int bsize, float mult, float* src, float *trg) {
    // Use shared memory to reduce global memory accesses
    extern __shared__ float shared_src[];
    
    // Calculate block and thread-specific indices
    int thread_id = threadIdx.x;
    int block_offset = blockIdx.x * bsize;
    
    // Load data into shared memory
    int data_idx = idx[blockIdx.x] * bsize + thread_id;
    if(thread_id < bsize) {
        shared_src[thread_id] = src[data_idx];
    }
    __syncthreads();

    // Perform the computation if within bounds
    int id = block_offset + thread_id;
    if(id < n * bsize) {
        trg[id] = shared_src[thread_id] * mult;
    }
}