#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_dense_to_sparse_subtract(int n, const unsigned int *idx, float *src, float *trg) {
    // Use a more efficient method to calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within bounds and directly update the target array
    if (id < n) {
        atomicAdd(&trg[idx[id]], -src[id]);
    }
}