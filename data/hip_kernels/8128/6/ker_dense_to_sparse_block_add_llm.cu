#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_dense_to_sparse_block_add(int n, const unsigned *idx, int bsize, float* src, float *trg) {
    // Calculate flattened thread id within grid
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Precompute idx base offset and src access to reduce redundant computations
    int baseOffset = (id / bsize) * bsize;
    
    // Ensure id is within bounds to prevent out-of-bounds access
    if (id < n * bsize) {
        // Use the precomputed offset for sparse index calculation
        atomicAdd(trg + idx[id / bsize] * bsize + id % bsize, src[id]);
    }
}