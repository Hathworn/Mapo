#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_sparse_to_dense_assign(int n, const unsigned int *idx, float *src, float *trg) {
    // Use shared memory to enhance data access efficiency
    extern __shared__ unsigned int s_idx[];

    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Load indices into shared memory
    if (id < n)
        s_idx[threadIdx.x] = idx[id];

    __syncthreads(); // Synchronize threads to ensure all indices are loaded

    // Make sure we do not go out of bounds
    if (id < n)
        trg[id] = src[s_idx[threadIdx.x]]; // Use shared memory for efficient data access
}