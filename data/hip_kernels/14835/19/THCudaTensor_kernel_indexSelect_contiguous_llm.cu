#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexSelect_contiguous( float *tensor, float *src, long stride, float *index, long idxSize)
{
    // Use shared memory for improved memory access patterns
    extern __shared__ float sharedSrc[];

    const int VT = 4;
    const int WARP_SIZE = 32;
    const int MAX_DIM_SIZE = 65535;

    for (int idx = blockIdx.x * blockDim.y + threadIdx.y; idx < idxSize; idx += blockDim.y * MAX_DIM_SIZE) {
        
        // Load index in register
        const int idxInBlock = threadIdx.y;
        const int srcIdx = ((int)index[idx] - 1) * stride;
        const int targetIdx = idx * stride;

        // First warp loads data into shared memory
        if (threadIdx.x < WARP_SIZE) {
            for (int startIdx = threadIdx.x; startIdx < stride; startIdx += WARP_SIZE) {
                sharedSrc[startIdx] = src[srcIdx + startIdx];
            }
        }
        __syncthreads();

        for (int startIdx = threadIdx.x + blockIdx.y * VT * WARP_SIZE; startIdx < stride; startIdx += VT * WARP_SIZE * MAX_DIM_SIZE) {
            #pragma unroll
            for (int i = 0; i < VT; i++) {
                const int featureIdx = startIdx + i * WARP_SIZE;
                if (featureIdx < stride) {
                    tensor[targetIdx + featureIdx] = sharedSrc[featureIdx];
                }
            }
        }
    }
}