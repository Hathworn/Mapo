#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexCopy(float *res, float *src, long* res_stride, float *index, long res_nDim, int dim, long idx_size, long src_size, long size_dim)
{
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    long flat_size = src_size / idx_size;

    if (thread_idx < flat_size)
    {
        for (int i = 0; i < idx_size; i++)
        {
            int leftover = thread_idx;
            int targetIdx = 0;
            int resIdx = 0;
            
            #pragma unroll // Unroll the loop for better performance
            for (int d = 0; d < res_nDim; d++)
            {
                long coeff = 0; // Moved declaration here for better scope control
                if (d < dim)
                {
                    long stride_d = res_stride[d] / size_dim;
                    coeff = leftover / stride_d;
                    leftover -= coeff * stride_d;
                    targetIdx += coeff * stride_d * idx_size;
                    resIdx += coeff * res_stride[d];
                }
                else if (d > dim)
                {
                    coeff = leftover / res_stride[d];
                    leftover -= coeff * res_stride[d];
                    targetIdx += coeff * res_stride[d];
                    resIdx += coeff * res_stride[d];
                }
            }
            // Combine index computation to minimize extraneous arithmetic
            int indexVal = static_cast<int>(index[i]) - 1;
            res[resIdx + indexVal * res_stride[dim]] = src[targetIdx + i * res_stride[dim]];
        }
    }
}