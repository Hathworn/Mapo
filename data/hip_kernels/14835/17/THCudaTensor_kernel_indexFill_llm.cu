#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexFill(float *tensor, long* stride, float *index, long src_nDim, int dim, long idx_size, long tensor_size, long size_dim, float val)
{
    // Calculate a unique index for each thread
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    // Calculate total number of elements each thread will process
    long flat_size = tensor_size / idx_size;

    if (thread_idx < flat_size)
    {
        for (int i = 0; i < idx_size; i++)
        {
            int leftover = thread_idx;
            int srcIdx = 0;
            // Combine the loops by removing redundant coefficient calculation
            for (int d = 0; d < src_nDim; d++)
            {
                if (d != dim)
                {
                    long divisor = (d < dim) ? (stride[d] / size_dim) : stride[d];
                    long coeff = leftover / divisor;
                    leftover -= coeff * divisor;
                    srcIdx += coeff * stride[d];
                }
            }
            // Update the tensor at calculated index
            tensor[srcIdx + (int)((index[i]) - 1) * stride[dim]] = val;
        }
    }
}