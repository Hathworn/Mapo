#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_copyUpperSymmetric(float *input, int n, int len)
{
    // Improvement: Use grid-stride loop for better utilization of GPU
    for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < len; idx += blockDim.x * gridDim.x) {
        const int r = idx % n; // Row index
        const int c = idx / n; // Column index
        if (r > c) {
            input[idx] = input[r*n + c]; // Copy upper symmetric matrix value
        }
    }
}