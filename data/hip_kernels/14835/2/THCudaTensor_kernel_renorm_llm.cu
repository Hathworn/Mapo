#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_renorm(float *data, const float value, const long size, const float maxnorm)
{
    __shared__ float buffer[32];
    long tx = threadIdx.x;
    long bx = blockIdx.x;
    long step = blockDim.x;
    float *row = data + size * bx;

    // Initialize buffer for each thread
    buffer[tx] = 0;

    // Compute partial norms
    for (long i = tx; i < size; i += step)
    {
        buffer[tx] += powf(fabsf(row[i]), value);
    }

    // Reduce within block
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
    {
        __syncthreads();
        if (tx < stride)
        {
            buffer[tx] += buffer[tx + stride];
        }
    }

    // Renormalize if necessary
    __syncthreads();
    if (tx == 0)
    {
        float norm = powf(buffer[0], 1 / value);
        if (norm > maxnorm)
        {
            float scale = maxnorm / (norm + 1e-7f);
            buffer[0] = scale; // Store scale factor for other threads
        }
        else
        {
            buffer[0] = 1.0f; // No scaling needed
        }
    }
    __syncthreads();

    float scale = buffer[0];
    if (scale < 1.0f)
    {
        for (long i = tx; i < size; i += step)
        {
            row[i] *= scale;
        }
    }
}