#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;

        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithoutReplacement(hiprandStateMtgp32* state, int totalSamples, int sample, float* dest, long distributions, int categories, float* origDist, float* normDistPrefixSum) {
    for (long curDistBase = blockIdx.x * blockDim.y; curDistBase < distributions; curDistBase += gridDim.x * blockDim.y) {
        long curDist = curDistBase + threadIdx.y;

        // Reduced divergence by using one random number for all threads in a block
        float r = hiprand_uniform(&state[blockIdx.x + threadIdx.y * blockDim.x]);

        if (threadIdx.x == 0 && curDist < distributions) {
            int choice = binarySearchForMultinomial(normDistPrefixSum + curDist * categories, categories, r);
            dest[curDist * totalSamples + sample] = (float) choice + 1.0f;
            origDist[curDist * categories + choice] = 0.0f; // Without replacement
        }
    }
}