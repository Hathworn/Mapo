#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;

        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        // No probability mass or precision problems; just return the first element
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithReplacement(hiprandStateMtgp32* state, int totalSamples, float* dest, long distributions, int categories, float* normDistPrefixSum) {
    // Cache the block index and grid dimensions for optimal access
    long curDist = blockIdx.x + blockIdx.y * gridDim.x; // Optimize distribution access
    
    for (; curDist < distributions; curDist += gridDim.x * gridDim.y) {
        for (int sampleBase = 0; sampleBase < totalSamples; sampleBase += blockDim.y) {
            int sample = sampleBase + threadIdx.y;

            // Use fewer threads for hiprand_uniform call; reduce divergence
            __shared__ float randVal;
            if (threadIdx.x == 0) {
                randVal = hiprand_uniform(&state[blockIdx.x]);
            }
            __syncthreads();

            // Using warp-wide binary search optimization
            if (sample < totalSamples) {
                int choice = binarySearchForMultinomial(
                    normDistPrefixSum + curDist * categories,
                    categories,
                    randVal);

                // Write result, 1-based index
                dest[curDist * totalSamples + sample] = static_cast<float>(choice + 1.0f);
            }
        }
    }
}