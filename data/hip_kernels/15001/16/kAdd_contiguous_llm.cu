#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a more efficient loop by leveraging single-thread execution per element
    if (idx < numEls) {
        dest[idx] = a[idx] + b[idx];
    }
}