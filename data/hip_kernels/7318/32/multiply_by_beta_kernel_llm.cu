#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiply_by_beta_kernel(float * input, float * output, float beta)
{
    // Optimize block and thread indexing by using a flat index
    unsigned long int id = blockIdx.z * (gridDim.y * gridDim.x) * blockDim.x * blockDim.y
                         + blockIdx.y * (gridDim.x * blockDim.x * blockDim.y)
                         + blockIdx.x * (blockDim.x * blockDim.y)
                         + threadIdx.z * (blockDim.x * blockDim.y)
                         + threadIdx.y * blockDim.x
                         + threadIdx.x;

    if (id < (c_Size.x * c_Size.y * c_Size.z))
    {
        output[id] = input[id] * beta; // Directly perform calculation for valid threads
    }
}