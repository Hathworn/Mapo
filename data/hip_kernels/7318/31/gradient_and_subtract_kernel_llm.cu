#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gradient_and_subtract_kernel(float * in, float * grad_x, float * grad_y, float * grad_z)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Exit if out of bounds
    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z) return;

    long int id = (k * c_Size.y + j) * c_Size.x + i;

    // Pre-calculate boundary checks and positions
    bool inside_x = (i < c_Size.x - 1);
    bool inside_y = (j < c_Size.y - 1);
    bool inside_z = (k < c_Size.z - 1);

    if (inside_x) {
        long int id_x = id + 1;
        grad_x[id] -= ((in[id_x] - in[id]) / c_Spacing.x); // Calculate gradient in x direction
    }
    if (inside_y) {
        long int id_y = id + c_Size.x;
        grad_y[id] -= ((in[id_y] - in[id]) / c_Spacing.y); // Calculate gradient in y direction
    }
    if (inside_z) {
        long int id_z = id + c_Size.x * c_Size.y;
        grad_z[id] -= ((in[id_z] - in[id]) / c_Spacing.z); // Calculate gradient in z direction
    }
}