#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_volume_to_constant(float *out, float value)
{
    // Use combined index calculation to reduce arithmetic operations
    unsigned int index = blockIdx.z * (gridDim.y * gridDim.x * blockDim.y * blockDim.x) +
                         blockIdx.y * (gridDim.x * blockDim.x) + 
                         blockIdx.x * blockDim.x +
                         threadIdx.z * (blockDim.y * blockDim.x) +
                         threadIdx.y * blockDim.x + 
                         threadIdx.x;

    unsigned int i = index % c_Size.x;
    unsigned int j = (index / c_Size.x) % c_Size.y;
    unsigned int k = index / (c_Size.x * c_Size.y);

    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
        return;

    out[index] = value; // Directly use calculated index
}