#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void magnitude_threshold_kernel(float * grad_x, float * grad_y, float * grad_z, float gamma)
{
    // Use fast math functions for efficiency
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Early exit to reduce register pressure
    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z) return;

    long int id = (k * c_Size.y + j) * c_Size.x + i;

    // Compute squared norm to avoid an extra sqrt operation
    float norm_sqr = grad_x[id] * grad_x[id] + grad_y[id] * grad_y[id] + grad_z[id] * grad_z[id];
    float gamma_sqr = gamma * gamma;

    if (norm_sqr > gamma_sqr)
    {
        // Avoid division by using reciprocal and multiplication
        float ratio = rsqrtf(norm_sqr) * gamma;
        grad_x[id] *= ratio;
        grad_y[id] *= ratio;
        grad_z[id] *= ratio;
    }
}