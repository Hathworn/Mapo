#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtract_kernel(float * in1, float * in2, float * out)
{
  // Calculate the unique global thread index corresponding to the 3D grid.
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

  // Calculate a linear index id.
  long int id = (k * c_Size.y + j) * c_Size.x + i;

  // Avoid redundant computation with early return
  // Check array bounds before accessing memory.
  if (i < c_Size.x && j < c_Size.y && k < c_Size.z) {
    out[id] = in1[id] - in2[id];
  }
}