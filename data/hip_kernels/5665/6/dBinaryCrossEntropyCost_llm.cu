#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dBinaryCrossEntropyCost(float* predictions, float* target, float* dY, int x) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds and perform calculations
    if (index < x) { 
        float pred = predictions[index];
        float tgt = target[index];
        
        // Optimize the calculation using temporary variables
        float inversePred = 1.0f - pred;
        float inverseTgt = 1.0f - tgt;
        
        dY[index] = -1.0f * (tgt / pred - inverseTgt / inversePred);
    }
}