#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reluForward(float* R, float* V, int x, int y) {
    // Use shared memory to minimize global memory accesses
    extern __shared__ float shared_V[];
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index < x * y) {
        // Load data into shared memory
        shared_V[threadIdx.x] = V[index];
        __syncthreads(); // Ensure all threads have written to shared memory

        // Perform ReLU operation
        R[index] = fmaxf(shared_V[threadIdx.x], 0);
    }
}