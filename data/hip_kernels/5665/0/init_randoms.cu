#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_randoms(unsigned int seed, hiprandState_t* states) {
int index = blockDim.x * blockIdx.x + threadIdx.x;

hiprand_init(seed, index, 0, &states[index]);
}