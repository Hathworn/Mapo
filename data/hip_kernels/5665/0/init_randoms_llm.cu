#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_randoms(unsigned int seed, hiprandState_t* states) {
    // Calculate unique thread index using direct calculation
    int index = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;

    // Initialize random state for each thread
    hiprand_init(seed, index, 0, &states[index]);
}