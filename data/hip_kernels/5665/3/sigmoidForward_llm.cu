#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float sigmoid(float x){
    return __frcp_rn(__fadd_rn(1, expf(-x))); // Use fast exp function for optimization
}

__global__ void sigmoidForward(float* R, float* V, int x, int y){
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int totalElements = x * y;  // Pre-compute total elements for efficiency
    if(index < totalElements)
        R[index] = sigmoid(V[index]);  // Optimize indexing within bounds
}