#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binaryCrossEntropyCost(float* cost, float* predictions, float* target, int size) {
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if index is within bounds
    if (index < size) {
        // Compute partial cost in registers for efficiency
        float pred = predictions[index];
        float tar = target[index];
        float partial_cost = tar * logf(1.0e-15 + pred) + (1.0f - tar) * logf(1.0e-15 + (1.0f - pred));
        
        // Atomic addition to avoid race conditions
        atomicAdd(cost, -partial_cost / size);
    }
}