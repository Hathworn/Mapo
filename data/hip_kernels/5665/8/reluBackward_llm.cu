#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reluBackward(float* dZ, float* top_diff, float* V, int x, int y) {
    // Calculate global index
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    // Prefetch index value of V
    float v_index = 0.0f;
    if (index < x * y) {
        v_index = V[index];
    }

    // Perform computation with register optimization
    if (v_index > 0) {
        dZ[index] = top_diff[index];
    } else {
        dZ[index] = 0;
    }
}