#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randoms(hiprandState_t* states, float* numbers, float lower, float higher) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    // Cache frequently accessed data in local variables
    float range = higher - lower;
    hiprandState_t localState = states[index];
    
    numbers[index] = lower + range * hiprand_uniform(&localState);

    // Store back the state
    states[index] = localState;
}