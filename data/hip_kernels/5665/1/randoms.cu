#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randoms(hiprandState_t* states, float* numbers, float lower, float higher) {
int index = blockDim.x * blockIdx.x + threadIdx.x;
numbers[index] = lower + (higher - lower) * hiprand_uniform(&states[index]);
}