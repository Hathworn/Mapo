#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float sigmoid_derivate(float x) {
    return __fmul_rn(x, __fsub_rn(1.0f, x));
}

__device__ float sigmoid(float x) {
    return __frcp_rn(__fadd_rn(1, exp(__fneg_rn(x))));
}

__global__ void sigmoidBackward(float* R, float* V, int x, int y) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Define stride to handle larger arrays
    for (int i = index; i < x * y; i += stride) {
        R[i] = sigmoid_derivate(V[i]);
    }
}