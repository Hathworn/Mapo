#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Calculate thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure only one thread performs the division
    if (idx == 0) {
        *accuracy /= N;
    }
}