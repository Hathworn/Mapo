#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void same_num_channels_div_kernel(const float *data_l, const float *data_r, float *result, unsigned total)
{
    // Calculate index accounting for each thread handling two data points
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < total) {
        // Improved indexing by eliminating repetitive multiplication
        unsigned dataIdx = 2 * idx;

        // Common denominator calculation optimization
        float denom = (data_r[dataIdx] * data_r[dataIdx] + data_r[dataIdx + 1] * data_r[dataIdx + 1]);

        // Ensure denominator is not zero
        if (denom != 0.0f) {
            result[dataIdx] = (data_l[dataIdx] * data_r[dataIdx] + data_l[dataIdx + 1] * data_r[dataIdx + 1]) / denom;
            result[dataIdx + 1] = (data_l[dataIdx + 1] * data_r[dataIdx] - data_l[dataIdx] * data_r[dataIdx + 1]) / denom;
        } else {
            result[dataIdx] = 0.0f;
            result[dataIdx + 1] = 0.0f;
        }
    }
}