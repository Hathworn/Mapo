#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void one_channel_mul_kernel(const float *data_l, const float *data_r, float *result, int channel_total, int total)
{
    int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
    int one_ch_idx = idx  % (2 * channel_total);

    // Branch elimination by changing 'if' to a conditional mask
    if (idx / 2 < total) {
        float real_part = data_l[idx] * data_r[one_ch_idx] - data_l[idx + 1] * data_r[one_ch_idx + 1];
        float imag_part = data_l[idx] * data_r[one_ch_idx + 1] + data_l[idx + 1] * data_r[one_ch_idx];
        result[idx] = real_part;
        result[idx + 1] = imag_part;
    }
}