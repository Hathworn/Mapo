#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void sum_channels(float *dest, const float *src, uint channels, uint num_channel_elem)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride to handle more elements and reduce unoptimized loop iteration
    for (uint i = idx; i < num_channel_elem; i += blockDim.x * gridDim.x) {
        float acc = 0;
        for (uint c = 0; c < channels; ++c) {
            acc += src[i + c * num_channel_elem];
        }
        dest[i] = acc;
    }
}