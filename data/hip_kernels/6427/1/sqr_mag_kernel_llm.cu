#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sqr_mag_kernel(const float *data, float *result, int total)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we're only accessing valid entries within the bound
    if (idx < total) {
        int dataIndex = 2 * idx;

        // Perform single read operation
        float real = data[dataIndex];
        float imag = data[dataIndex + 1];

        // Calculate squared magnitude
        result[dataIndex] = real * real + imag * imag;
        // Reset imaginary part to zero
        result[dataIndex + 1] = 0;
    }
}