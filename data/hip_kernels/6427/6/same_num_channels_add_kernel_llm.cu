#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void same_num_channels_add_kernel(const float *data_l, const float *data_r, float *result, int total)
{
    // Calculate global thread index, optimize by using 1-based index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure array boundaries are respected
    if (idx < total) {
        int base = 2 * idx; // Compute base index for channel data
        result[base] = data_l[base] + data_r[base]; // Perform addition for channel 0
        result[base + 1] = data_l[base + 1] + data_r[base + 1]; // Perform addition for channel 1
    }
}