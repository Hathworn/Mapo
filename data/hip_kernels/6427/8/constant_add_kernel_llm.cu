#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constant_add_kernel(const float *data_l, float constant, float *result, int total)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Access two elements per thread efficiently
    int base_idx = 2 * idx;

    if (idx < total) { // Use idx directly for proper boundary check
        result[base_idx] = data_l[base_idx] + constant;
        result[base_idx + 1] = data_l[base_idx + 1];
    }
}