#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sqr_norm_kernel(const float *in, float *block_res, int total)
{
    extern __shared__ float sdata[];
    int in_idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
    int i = threadIdx.x;
    unsigned ins = blockDim.x;

    // Improved boundary check with conditional operator
    sdata[i] = (in_idx < total * 2) ? (in[in_idx] * in[in_idx] + in[in_idx + 1] * in[in_idx + 1]) : 0;

    // Optimized loop with power-of-two assumption for block size
    for (unsigned stride = ins / 2; stride > 0; stride /= 2) {
        __syncthreads();
        if (i < stride)
            sdata[i] += sdata[i + stride];
    }

    // Store result from the block
    if (i == 0)
        block_res[blockIdx.x] = sdata[0];
}