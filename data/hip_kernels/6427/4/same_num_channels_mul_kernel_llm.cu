#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void same_num_channels_mul_kernel(const float *data_l, const float *data_r, float *result, int total)
{
    int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);

    // Check if idx is valid within bounds
    if (idx < 2 * total) {
        float l0 = data_l[idx];
        float l1 = data_l[idx + 1];
        float r0 = data_r[idx];
        float r1 = data_r[idx + 1];

        // Perform complex number multiplication
        result[idx] = l0 * r0 - l1 * r1;
        result[idx + 1] = l0 * r1 + l1 * r0;
    }
}