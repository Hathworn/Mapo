#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constant_mul_kernel(const float *data_l, float constant, float *result, int total)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process two elements per thread if within bounds, avoid unnecessary division
    int offset = idx * 2;
    if (offset < total * 2) {
        result[offset] = data_l[offset] * constant;
        result[offset + 1] = data_l[offset + 1] * constant;
    }
}