#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conj_kernel(const float *data, float *result, int total)
{
    // Use a single thread per complex number to optimize memory access
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < total) {
        int offset = 2 * idx; // Calculate offset for real and imaginary components
        result[offset] = data[offset];
        result[offset + 1] = -data[offset + 1];
    }
}