#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Dot(float *a, float *b, float *c)
{
    __shared__ float cache[ThreadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float temp = 0.0f;

    // Loop unrolling for better performance
    int stride = blockDim.x * gridDim.x;
    while (tid < N - 3) {
        temp += a[tid] * b[tid] 
              + a[tid + 1] * b[tid + 1] 
              + a[tid + 2] * b[tid + 2] 
              + a[tid + 3] * b[tid + 3];
        tid += stride;
    }

    // Remaining elements
    for (; tid < N; tid += stride) {
        temp += a[tid] * b[tid];
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    // Improved parallel reduction
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}