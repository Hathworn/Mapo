#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Zero_Dot(float *a, float *b, float *c)
{
    __shared__ float cache[ThreadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float temp = 0;

    // Optimize data parallelism by unrolling the loop
    const int gridSize = blockDim.x * gridDim.x;
    while (tid < N) {
        temp += a[tid] * b[tid];
        if (tid + gridSize < N) temp += a[tid + gridSize] * b[tid + gridSize];
        tid += gridSize * 2; // Increment by twice the gridSize
    }
    
    cache[cacheIndex] = temp;
    __syncthreads();

    // Optimizing reduction using loop unrolling technique
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}