#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZE (100 * 1024 * 1024)

__global__ void histo_kernel(unsigned char *buffer, int size, unsigned int *histo)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory for histogram to reduce global memory accesses
    __shared__ unsigned int sharedHisto[256];
    if (threadIdx.x < 256)
    {
        sharedHisto[threadIdx.x] = 0;
    }
    __syncthreads();

    while (i < size)
    {
        atomicAdd(&sharedHisto[buffer[i]], 1); // Accumulate count in shared memory
        i += stride;
    }
    __syncthreads();

    // Aggregate shared memory histogram into global memory
    if (threadIdx.x < 256)
    {
        atomicAdd(&histo[threadIdx.x], sharedHisto[threadIdx.x]);
    }
}