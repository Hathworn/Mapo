#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZE (100 * 1024 * 1024)

__global__ void histo_kernel_optimization(unsigned char *buffer, int size, unsigned int *histo)
{
    // Use shared memory to prevent bank conflicts and achieve better performance
    __shared__ unsigned int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Accumulate histogram counts into shared memory
    while (i < size)
    {
        atomicAdd(&temp[buffer[i]], 1);
        i += stride;
    }
    __syncthreads();

    // Use atomic operations to update the global histogram
    atomicAdd(&histo[threadIdx.x], temp[threadIdx.x]);
}