#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(int *d_num_steps, unsigned long long *d_fact, double *d_out) {
    __shared__ double local_d_out[256]; // Use shared memory for partial results

    int idx = threadIdx.x;
    int num_steps = *d_num_steps;
    local_d_out[idx] = 0; // Initialize shared memory

    for (int k = idx + 1; k < num_steps; k += blockDim.x) {
        local_d_out[idx] += (double) k * 0.5 / (double) d_fact[k - 1];
    }

    d_out[idx] = local_d_out[idx]; // Write back to global memory
}