#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(float *d_out, float *d_in) {
    // Use blockIdx.x and blockDim.x for multi-block capability
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid invalid memory access
    if (idx < N) { // Assume N is defined as the size of d_in and d_out
        float f = d_in[idx];
        d_out[idx] = f * f;
    }
}