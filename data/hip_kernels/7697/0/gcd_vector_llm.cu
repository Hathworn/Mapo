#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
int threads;

__global__ void gcd_vector(int *d_out, int integer_m) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Calculate global thread index
    int stride = blockDim.x * gridDim.x;  // Calculate stride
    int count = 0;  // Local count variable

    for (int i = idx; i < integer_m; i += stride) {  // Iterate with global stride
        int u = i, v = integer_m;
        while (v != 0) {
            int r = u % v;
            u = v;
            v = r;
        }
        if (u == 1) {
            count++;  // Count GCD equals 1
        }
    }

    atomicAdd(&d_out[0], count);  // Atomic addition to global output
}