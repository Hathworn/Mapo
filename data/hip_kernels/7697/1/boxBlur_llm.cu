#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void boxBlur(uchar4 *d_img, uchar4 *d_blurred, int numRows, int numCols, int k){

    // Calculate global thread ID
    int threadX = blockIdx.x * blockDim.x + threadIdx.x;
    int threadY = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the current thread is within bounds
    if (threadX >= numRows || threadY >= numCols) return;

    int pixel = numCols * threadX + threadY;
    int cornerX = threadX - (k / 2);
    int cornerY = threadY - (k / 2);

    int sumR = 0, sumG = 0, sumB = 0, count = 0;

    // Iterate over the kernel
    for (int i = 0; i < k; ++i) {
        for (int j = 0; j < k; ++j) {
            int currentX = cornerX + i;
            int currentY = cornerY + j;

            // Check if within bounds
            if (currentX >= 0 && currentX < numRows && currentY >= 0 && currentY < numCols) {
                int index = numCols * currentX + currentY;
                uchar4 currentColor = d_img[index];
                sumR += (int)currentColor.x;
                sumG += (int)currentColor.y;
                sumB += (int)currentColor.z;
                count++;
            }
        }
    }

    // Average color values
    uchar4 blurredColor;
    blurredColor.x = count ? (sumR / count) : d_img[pixel].x;
    blurredColor.y = count ? (sumG / count) : d_img[pixel].y;
    blurredColor.z = count ? (sumB / count) : d_img[pixel].z;

    // Write the resulting color to the blurred image
    d_blurred[pixel] = blurredColor;
}