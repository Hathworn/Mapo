#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMul(float *Md, float *Nd, float *Pd, const int WIDTH) {
    // Calculate thread id
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

    // Temporary variable for accumulating the result
    float Pvalue = 0;

    // Optimize loop by reducing global memory accesses
    for (int k = 0; k < WIDTH; k++) {
        Pvalue += Md[row * WIDTH + k] * Nd[k * WIDTH + col];
    }

    // Write the result to global memory
    Pd[row * WIDTH + col] = Pvalue;
}