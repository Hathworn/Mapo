#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpy(double *y, double *a, double *x, int m, int n) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];

    if (bid < m) {
        // Calculate index for accessing shared memory.
        int index = tid;
        
        if (index < n) {
            // Perform multiplication for each element.
            dots_s[index] = a[bid * n + index] * x[index];
        }
        __syncthreads();

        // Use parallel reduction for summation.
        for (int offset = n / 2; offset > 0; offset >>= 1) {
            if (index < offset) {
                dots_s[index] += dots_s[index + offset];
            }
            __syncthreads();
        }

        // Store the result of the reduction.
        if (index == 0) {
            y[bid] = dots_s[0];
        }
    }
}