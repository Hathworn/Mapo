#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void suma(int a, int b, int *c) {
    // Use shared memory for faster access
    __shared__ int temp;
    temp = a + b;

    // Use atomic operation to safely write the result from multiple threads
    if (threadIdx.x == 0) {
        atomicAdd(c, temp);
    }
}