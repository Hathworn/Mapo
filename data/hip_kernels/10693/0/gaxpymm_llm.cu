#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpymm(double *y, double *a, double *b, int m, int n, int p) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];

    if (bid < m) {
        // Read input data into shared memory
        double temp_sum = 0.0;
        for (int c = 0; c < p; c++) {
            if (tid < n) {
                dots_s[tid * p + c] = a[bid * n + tid] * b[tid * p + c];
            }
        }
        __syncthreads();

        // Accumulate results with parallel reduction
        for (int c = 0; c < p; c++) {
            for (int stride = 1; stride < n; stride *= 2) {
                if (tid % (2 * stride) == 0 && tid + stride < n) {
                    dots_s[tid * p + c] += dots_s[(tid + stride) * p + c];
                }
                __syncthreads();
            }
            // Write result to global memory
            if (tid == 0) {
                y[bid * p + c] = dots_s[c];
            }
        }
    }
}