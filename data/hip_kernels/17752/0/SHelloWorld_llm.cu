#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SHelloWorld()
{
    // Calculate global thread index for better control
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("SubHelloWorld from %d-%d\n", blockIdx.x, globalIdx);
}