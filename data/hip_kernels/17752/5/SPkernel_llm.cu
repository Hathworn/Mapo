#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SPkernel(int k, int m, int n, float* searchPoints, float* referencePoints, int* indices)
{
    extern __shared__ float sharedReferencePoints[]; // Shared memory for reference points
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int threadLane = threadIdx.x;
    
    if (tid < m) {
        float minSquareSum = FLT_MAX; // Use max float value for comparison
        int minIndex = -1;

        // Iterate over all reference points in blocks
        for (int nStart = 0; nStart < n; nStart += blockDim.x) {
            // Load reference points into shared memory
            if (nStart + threadLane < n) {
                for (int offset = 0; offset < k; ++offset) {
                    sharedReferencePoints[threadLane * k + offset] = referencePoints[(nStart + threadLane) * k + offset];
                }
            }
            __syncthreads(); // Ensure all threads have loaded data

            // Compare current search point with reference points in shared memory
            for (int nInd = 0; nInd < blockDim.x && (nStart + nInd) < n; ++nInd) {
                float squareSum = 0;
                for (int kInd = 0; kInd < k; kInd++) {
                    float diff = searchPoints[tid * k + kInd] - sharedReferencePoints[nInd * k + kInd];
                    squareSum += (diff * diff);
                }
                if (squareSum < minSquareSum) {
                    minSquareSum = squareSum;
                    minIndex = nStart + nInd;
                }
            }
            __syncthreads(); // Ensure all thread calculations are done before next load
        }
        indices[tid] = minIndex;
    }
}