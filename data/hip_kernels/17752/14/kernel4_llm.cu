#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel4(int k, int n, int bias, float* searchPoints, float* referencePoints, float* dist)
{
    // Cache tid to avoid recalculating it repeatedly
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure this thread is within bounds
    if (tid < n) {
        // Use register to store square sum for faster access
        float squareSum = 0;

        // Use constant memory for bias multiplication to enhance memory access speed
        const int baseIndexSearch = k * bias;
        const int baseIndexRef = k * tid;
        
        // Iterate over dimensions using optimized memory access
        for (int i = 0; i < k; i++) {
            float diff = searchPoints[baseIndexSearch + i] - referencePoints[baseIndexRef + i];
            squareSum += diff * diff;
        }
        
        // Store the result in the output array
        dist[bias * n + tid] = squareSum;
    }
}