#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SPkernel(int k, int m, int n, float* searchPoints, float* referencePoints, int* indices)
{
    // Calculate global thread ID
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Check if tid is within the range
    if (tid < m) {
        int minIndex = -1;
        float minSquareSum = FLT_MAX; // Initialize to max float value for comparison
        
        // Iterate over all reference points
        for (int nInd = 0; nInd < n; nInd++) {
            float squareSum = 0.0f;
            
            // Compute the square of the differences
            for (int kInd = 0; kInd < k; kInd++) {
                float diff = searchPoints[k * tid + kInd] - referencePoints[k * nInd + kInd];
                squareSum += (diff * diff);
            }
            
            // Update minimum square sum and index
            if (squareSum < minSquareSum) {
                minSquareSum = squareSum;
                minIndex = nInd;
            }
        }
        indices[tid] = minIndex; // Store the index of the nearest reference point
    }
}