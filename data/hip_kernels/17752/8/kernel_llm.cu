#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(int k, int n, float* sub_searchPoints, float* referencePoints, float* dist)
{
    float diff, squareSum = 0;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n) {
        float* refPoint = &referencePoints[k * tid]; // Cache memory access
        for (int i = 0; i < k; i++) {
            diff = sub_searchPoints[i] - refPoint[i];
            squareSum += (diff * diff);
        }
        dist[tid] = squareSum;
    }
}