#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel2(int k, int n, float* sub_searchPoints, float* referencePoints, float* dist)
{
    // Calculate unique thread identifier
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure within bounds
    if (tid < n) {
        float squareSum = 0;

        // Use shared memory for sub_searchPoints
        __shared__ float shared_searchPoints[1024]; // Assume max k is 1024
        if (threadIdx.x < k) {
            shared_searchPoints[threadIdx.x] = sub_searchPoints[threadIdx.x];
        }
        __syncthreads(); // Synchronize threads

        for (int i = 0; i < k; i++) {
            float diff = shared_searchPoints[i] - referencePoints[k * tid + i];
            squareSum += (diff * diff);
        }
        dist[tid] = squareSum;
    }
}