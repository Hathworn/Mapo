#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RPkernel(int k, int n, float* sub_searchPoints, float* referencePoints, float* dist)
{
    __shared__ float s_sub_searchPoints[1024]; // Use shared memory for better locality
    float diff, squareSum;

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure threads within the block load `sub_searchPoints` into shared memory
    if (threadIdx.x < k) {
        s_sub_searchPoints[threadIdx.x] = sub_searchPoints[threadIdx.x];
    }
    __syncthreads(); // Synchronize to ensure all data is loaded before computation

    if (tid < n) {
        squareSum = 0.0f;
        for (int i = 0; i < k; i++) {
            diff = s_sub_searchPoints[i] - referencePoints[k * tid + i];
            squareSum += (diff * diff);
        }
        dist[tid] = squareSum;
    }
}