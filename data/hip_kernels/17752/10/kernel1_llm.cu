#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel1(int k, int m, int n, float* searchPoints, float* referencePoints, int* indices)
{
    extern __shared__ float sharedSearchPoints[]; // Use shared memory for search points
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int localTid = threadIdx.x;
    
    // Each block loads a subset of searchPoints into shared memory
    if (tid < m) {
        for (int i = localTid; i < k; i += blockDim.x) {
            sharedSearchPoints[i] = searchPoints[k * tid + i];
        }
    }
    __syncthreads();
   
    if (tid < m) {
        float minSquareSum = INFINITY;
        int minIndex = -1;

        // Iterate over all reference points
        for (int nInd = 0; nInd < n; nInd++) {
            float squareSum = 0.0f;
            // Compute difference using shared memory
            for (int kInd = 0; kInd < k; kInd++) {
                float diff = sharedSearchPoints[kInd] - referencePoints[k * nInd + kInd];
                squareSum += (diff * diff);
            }
            // Update the minimum square sum and index
            if (squareSum < minSquareSum) {
                minSquareSum = squareSum;
                minIndex = nInd;
            }
        }
        indices[tid] = minIndex;
    }
}