#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RPkernel(int k, int n, float* sub_searchPoints, float* referencePoints, float* dist)
{
    extern __shared__ float s_searchPoints[]; // Shared memory for sub_searchPoints
    
    // Load sub_searchPoints to shared memory
    if (threadIdx.x < k) {
        s_searchPoints[threadIdx.x] = sub_searchPoints[threadIdx.x];
    }
    __syncthreads(); // Ensure all threads have loaded sub_searchPoints

    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n) {
        float diff, squareSum = 0.0f;

        // Loop unrolling for improved performance
        for (int i = 0; i < k; i += 4) {
            diff = s_searchPoints[i] - referencePoints[k * tid + i];
            squareSum += diff * diff;
            if (i + 1 < k) {
                diff = s_searchPoints[i + 1] - referencePoints[k * tid + i + 1];
                squareSum += diff * diff;
            }
            if (i + 2 < k) {
                diff = s_searchPoints[i + 2] - referencePoints[k * tid + i + 2];
                squareSum += diff * diff;
            }
            if (i + 3 < k) {
                diff = s_searchPoints[i + 3] - referencePoints[k * tid + i + 3];
                squareSum += diff * diff;
            }
        }
        dist[tid] = squareSum;
    }
}