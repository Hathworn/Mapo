#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(int k, int n, float* sub_searchPoints, float* referencePoints, float* dist)
{
    // Calculate thread ID
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure the thread index is within bounds
    if (tid < n) {
        float squareSum = 0;
        // Unrolling loop to improve performance
        for (int i = 0; i < k; i += 4) {
            float diff1 = sub_searchPoints[i] - referencePoints[k * tid + i];
            float diff2 = sub_searchPoints[i + 1] - referencePoints[k * tid + i + 1];
            float diff3 = sub_searchPoints[i + 2] - referencePoints[k * tid + i + 2];
            float diff4 = sub_searchPoints[i + 3] - referencePoints[k * tid + i + 3];
            squareSum += (diff1 * diff1 + diff2 * diff2 + diff3 * diff3 + diff4 * diff4);
        }
        // Store result
        dist[tid] = squareSum;
    }
}