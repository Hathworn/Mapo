#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void kernel(int k, int m, int n, float* searchPoints, float* referencePoints, int* indices)
{
    // Calculate thread ID
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure thread is within bounds
    if (tid < m) {
        float minSquareSum = FLT_MAX; // Use maximum float as initial min
        int minIndex = -1;

        // Iterate over all reference points
        for (int nInd = 0; nInd < n; nInd++) {
            float squareSum = 0.0f;

            // Vectorized computation using shared memory may be applied in real scenarios, if applicable
            for (int kInd = 0; kInd < k; kInd++) {
                float diff = searchPoints[k * tid + kInd] - referencePoints[k * nInd + kInd];
                squareSum += (diff * diff);
            }

            // Find minimum distance
            if (squareSum < minSquareSum) {
                minSquareSum = squareSum;
                minIndex = nInd;
            }
        }

        // Store the index of the closest reference point
        indices[tid] = minIndex;
    }
}