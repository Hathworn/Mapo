#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];

    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    const int b = blockDim.x;
    int p = y * pitch + x;

    // Load the initial values into shared memory
    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        minvals[tx] = FLT_MAX;
        maxvals[tx] = -FLT_MAX;
    }

    // Process each row
    for (int ty = 1; ty < 16 && (y + ty) < height; ty++) {
        p += pitch;
        if (x < width) {
            float val = d_Data[p];
            minvals[tx] = fminf(minvals[tx], val);
            maxvals[tx] = fmaxf(maxvals[tx], val);
        }
    }
    __syncthreads();

    // Reduce within the block
    for (int d = b >> 1; d > 0; d >>= 1) {
        if (tx < d) {
            minvals[tx] = fminf(minvals[tx], minvals[tx + d]);
            maxvals[tx] = fmaxf(maxvals[tx], maxvals[tx + d]);
        }
        __syncthreads();
    }

    // Write the result for this block
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}