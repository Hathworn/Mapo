#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height) {
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    const int b = blockDim.x;
    int p = y * width + x;
    
    // Load initial values and handle boundary conditions
    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        float val = -1.0e10; // Use a very large negative number
        minvals[tx] = val;
        maxvals[tx] = val;
    }
    
    // Update min and max values within the block
    for (int ty = 1; ty < 16; ty++) {
        p += width;
        if (x < width && (y + ty) < height) {
            float val = d_Data[p];
            if (val < minvals[tx]) minvals[tx] = val;
            if (val > maxvals[tx]) maxvals[tx] = val;
        }
    }
    
    __syncthreads();
    
    // Reduction to find block minimum and maximum values
    for (int d = b >> 1; d > 0; d >>= 1) {
        if (tx < d) {
            if (minvals[tx + d] < minvals[tx]) minvals[tx] = minvals[tx + d];
            if (maxvals[tx + d] > maxvals[tx]) maxvals[tx] = maxvals[tx + d];
        }
        __syncthreads();
    }
    
    // Store results in global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}