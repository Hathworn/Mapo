#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Calculate the thread's unique index within the 3D grid
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate the size of a single plane
    int sz = height * pitch;

    // Calculate the linear index for the current thread
    int p = threadIdx.z * sz + y * pitch + x;

    // Boundary check and perform the operation within valid data range
    if (x < width && y < height) {
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
}
```
