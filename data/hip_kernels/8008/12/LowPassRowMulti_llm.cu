#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LowPassRowMulti(float *d_Result, float *d_Data, int width, int pitch, int height) {
    __shared__ float data[CONVROW_W + 2 * RADIUS];
    const int tx = threadIdx.x;
    const int block = blockIdx.x / (NUM_SCALES + 3);
    const int scale = blockIdx.x - (NUM_SCALES + 3) * block;
    const int xout = block * CONVROW_W + tx;
    const int loadPos = xout - RADIUS;
    const int yptr = blockIdx.y * pitch;
    const int writePos = yptr + height * pitch * scale + xout;
    float *kernel = d_Kernel + scale * 16;

    // Load data with boundary checks
    if (loadPos < 0)
        data[tx] = d_Data[yptr];
    else if (loadPos >= width)
        data[tx] = d_Data[yptr + width - 1];
    else
        data[tx] = d_Data[yptr + loadPos];

    __syncthreads();

    if (xout < width && tx < CONVROW_W) {
        // Optimize memory access and computation
        float val = data[tx + 4] * kernel[4];
        for (int k = 0; k < RADIUS; ++k) {
            val += (data[tx + k] + data[tx + 8 - k]) * kernel[k];
        }
        d_Result[writePos] = val;
    }
    __syncthreads();
}