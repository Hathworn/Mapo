#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    // Use shared memory to optimize access to minmax array
    __shared__ unsigned int sdata[32][16];

    const int tx = threadIdx.x;
    const int bidx = blockIdx.x;
    const int stride = gridDim.x;

    int numPts = 0;

    for (int y = bidx; y < h / 32; y += stride) {
        for (int x = 0; x < w; x += 16) {
            sdata[threadIdx.y][tx] = minmax[(y * 32 + threadIdx.y) * w + x + tx];

            __syncthreads();

            unsigned int val = sdata[threadIdx.y][tx];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }

            __syncthreads();
        }
    }
}