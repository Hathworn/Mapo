#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    // Obtain thread ID
    const int tx = threadIdx.x;

    // Shared memory for minmax values per thread block
    __shared__ unsigned int sh_minmax[32 * (16 / 32)];

    // Block-level shared numPts for atomic updates
    __shared__ int sh_numPts;
    if (tx == 0) sh_numPts = 0;
    __syncthreads();

    // Iterate over rows in steps of 32
    for (int y = 0; y < h / 32; y++) {
        for (int x = 0; x < w; x += 16) {

            // Load minmax values into shared memory
            if (x+tx < w) {
                sh_minmax[tx] = minmax[y * w + x + tx];
            }
            else {
                sh_minmax[tx] = 0;
            }
            __syncthreads();

            unsigned int val = sh_minmax[tx];

            if (val) {
                // Check each bit in val
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1) {
                        // Use atomic add to safely increment global numPts
                        int index = atomicAdd(&sh_numPts, 1);
                        if (index < maxPts) {
                            ptrs[16 * index + tx] = (y * 32 + k) * w + x + tx;
                        }
                    }
                    val >>= 1;
                }
            }
            __syncthreads();
        }
    }
}