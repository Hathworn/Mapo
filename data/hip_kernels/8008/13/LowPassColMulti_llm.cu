#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LowPassColMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    __shared__ float data[CONVCOL_W*(CONVCOL_H + 2*RADIUS)];
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int block = blockIdx.x/(NUM_SCALES+3);
    const int scale = blockIdx.x - (NUM_SCALES+3)*block;
    const int miny = blockIdx.y*CONVCOL_H;
    const int maxy = min(miny + CONVCOL_H, height) - 1;
    const int totStart = miny - RADIUS;
    const int totEnd = maxy + RADIUS;
    const int colStart = block*CONVCOL_W + tx;
    const int colEnd = colStart + (height-1)*pitch;
    const int sStep = CONVCOL_W*CONVCOL_S;
    const int gStep = pitch*CONVCOL_S;
    float *kernel = d_Kernel + scale*16;
    const int size = pitch*height*scale;
    d_Result += size;
    d_Data += size;

    if (colStart < width) {
        float *sdata = data + ty*CONVCOL_W + tx;
        int gPos = colStart + (totStart + ty) * pitch;
        for (int y = totStart + ty; y <= totEnd; y += blockDim.y) {
            sdata[0] = (y < 0) ? d_Data[colStart] : (y >= height) ? d_Data[colEnd] : d_Data[gPos];
            sdata += sStep;
            gPos += gStep;
        }
    }
    __syncthreads();

    if (colStart < width) {
        float *sdata = data + ty*CONVCOL_W + tx;
        int gPos = colStart + (miny + ty) * pitch;
        for (int y = miny + ty; y <= maxy; y += blockDim.y) {
            float res = 0.0f;
            #pragma unroll
            for (int i = 0; i < 5; ++i) {
                res += sdata[i*CONVCOL_W] * kernel[i] + sdata[(8-i)*CONVCOL_W] * kernel[i];
            }
            d_Result[gPos] = res;
            sdata += sStep;
            gPos += gStep;
        }
    }
}