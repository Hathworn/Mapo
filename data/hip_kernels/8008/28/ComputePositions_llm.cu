#include "hip/hip_runtime.h"
```c++
#include "hip/hip_runtime.h"
#include "includes.h"

#define POSBLK_SIZE 64

__global__ void ComputePositions(float *g_Data1, float *g_Data2, float *g_Data3, int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
    int i = blockIdx.x * POSBLK_SIZE + threadIdx.x;
    if (i >= numPts)
        return;

    int p = d_Ptrs[i];
    
    // Load neighbor elements into shared memory for data reuse and reduced global memory accesses
    __shared__ float sharedData[POSBLK_SIZE][8];

    if (threadIdx.x < POSBLK_SIZE) 
    {
        sharedData[threadIdx.x][0] = g_Data2[p];
        sharedData[threadIdx.x][1] = g_Data2[p - 1];
        sharedData[threadIdx.x][2] = g_Data2[p + 1];
        sharedData[threadIdx.x][3] = g_Data2[p - w];
        sharedData[threadIdx.x][4] = g_Data2[p + w];
        sharedData[threadIdx.x][5] = g_Data3[p];
        sharedData[threadIdx.x][6] = g_Data1[p];
        sharedData[threadIdx.x][7] = 2.0f * sharedData[threadIdx.x][0] - sharedData[threadIdx.x][1] - sharedData[threadIdx.x][2];
    }
    
    __syncthreads(); // Ensure all threads have loaded their data into shared memory

    float val[7];
    for (int j = 0; j < 7; j++)
    {
        val[j] = sharedData[threadIdx.x][j];
    }

    float dx = 0.5f * (val[2] - val[1]);
    float dxx = sharedData[threadIdx.x][7];
    float dy = 0.5f * (val[4] - val[3]);
    float dyy = 2.0f * val[0] - val[3] - val[4];
    float ds = 0.5f * (val[6] - val[5]);
    float dss = 2.0f * val[0] - val[5] - val[6];
    float dxy = 0.25f * (g_Data2[p + w + 1] + g_Data2[p - w - 1] - g_Data2[p - w + 1] - g_Data2[p + w - 1]);
    float dxs = 0.25f * (g_Data3[p + 1] + g_Data1[p - 1] - g_Data1[p + 1] - g_Data3[p - 1]);
    float dys = 0.25f * (g_Data3[p + w] + g_Data1[p - w] - g_Data3[p - w] - g_Data1[p + w]);

    float idxx = dyy * dss - dys * dys;
    float idxy = dys * dxs - dxy * dss;
    float idxs = dxy * dys - dyy * dxs;
    float idyy = dxx * dss - dxs * dxs;
    float idys = dxy * dxs - dxx * dys;
    float idss = dxx * dyy - dxy * dxy;
    float det = idxx * dxx + idxy * dxy + idxs * dxs;
    float idet = 1.0f / det;

    float pdx = idet * (idxx * dx + idxy * dy + idxs * ds);
    float pdy = idet * (idxy * dx + idyy * dy + idys * ds);
    float pds = idet * (idxs * dx + idys * dy + idss * ds);

    if (pdx < -0.5f || pdx > 0.5f || pdy < -0.5f || pdy > 0.5f || pds < -0.5f || pds > 0.5f) 
    {
        pdx = __fdividef(dx, dxx);
        pdy = __fdividef(dy, dyy);
        pds = __fdividef(ds, dss);
    }

    float dval = 0.5f * (dx * pdx + dy * pdy + ds * pds);
    d_Sift[i + 0 * maxPts] = (p % w) + pdx;
    d_Sift[i + 1 * maxPts] = (p / w) + pdy;
    d_Sift[i + 2 * maxPts] = d_ConstantA[0] * exp2f(pds * d_ConstantB[0]);
    d_Sift[i + 3 * maxPts] = val[0] + dval;

    float tra = dxx + dyy;
    det = dxx * dyy - dxy * dxy;
    d_Sift[i + 4 * maxPts] = __fdividef(tra * tra, det);
}