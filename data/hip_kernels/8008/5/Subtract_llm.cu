#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    // Compute the global thread coordinates
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate the 1D index from 2D coordinates
    int p = y * pitch + x;

    // Boundary check
    if (x < width && y < height) {
        // Perform element-wise subtraction
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}