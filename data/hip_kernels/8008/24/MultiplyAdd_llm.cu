#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Calculate x and y indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Calculate linear index
    int p = y * width + x;

    // Ensure within bounds before computation
    if (x < width && y < height) {
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
}