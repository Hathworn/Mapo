#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAddKernel(float *A, float *B, float *C, int n) {
    // Precompute thread index to improve readability
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use branch-less instruction to reduce control divergence
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}