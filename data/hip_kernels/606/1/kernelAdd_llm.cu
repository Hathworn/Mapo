#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelAdd(float *dvalues, int numOperations, int firstInd, int nextColInd)
{
    int vi = firstInd + blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread operation is within bounds
    if (vi < nextColInd) {
        float value = dvalues[vi]; // Cache data value to register for fast access
        for (int j = 0; j < numOperations; ++j) {
            value *= value + value * value; // Perform computation using cached value
        }
        dvalues[vi] = value; // Write back to global memory
    }
}