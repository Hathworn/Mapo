#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//=============================================================================
// FILE:   mytoy.cu
// AUTHORS: Raul Segura & Manuel Ujaldon (copyright 2014)
// Look for the string "MU" whenever Manuel suggests you to introduce changes
// Feel free to change some other parts of the code too (at your own risk)
//=============================================================================

//=============================================================================
// CUDA functions.
//=============================================================================

//Error handler for CUDA functions.
__global__ void kernelAdd(float *dvalues, int numOperations, int firstInd, int nextColInd)
{
    int vi = firstInd + blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize: Check boundary condition outside the loop
    if (vi < nextColInd) {
        float tempValue = dvalues[vi];  // Cache value to reduce repeated memory access
        for (int j = 0; j < numOperations; ++j) {
            tempValue *= tempValue + tempValue * tempValue; // Compute using cached value
        }
        dvalues[vi] = tempValue;  // Write back result after loop
    }
}