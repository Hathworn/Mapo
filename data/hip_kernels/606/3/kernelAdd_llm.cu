#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelAdd(float *dvalues, int numOperations, int firstInd, int nextColInd)
{
    int vi = firstInd + blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce global memory accesses
    __shared__ float sdata[256];  // Assuming a maximum of 256 threads per block
    if (vi < nextColInd) {
        sdata[threadIdx.x] = dvalues[vi];
        __syncthreads();  // Ensure all threads have loaded their data

        // Loop unrolling to reduce iteration overhead
        for (int j = 0; j < numOperations; j += 4) {
            sdata[threadIdx.x] *= sdata[threadIdx.x] + sdata[threadIdx.x] * sdata[threadIdx.x];
            sdata[threadIdx.x] *= sdata[threadIdx.x] + sdata[threadIdx.x] * sdata[threadIdx.x];
            sdata[threadIdx.x] *= sdata[threadIdx.x] + sdata[threadIdx.x] * sdata[threadIdx.x];
            sdata[threadIdx.x] *= sdata[threadIdx.x] + sdata[threadIdx.x] * sdata[threadIdx.x];
        }

        // Store results back to global memory
        dvalues[vi] = sdata[threadIdx.x];
    }
}