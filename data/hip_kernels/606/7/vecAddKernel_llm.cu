#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAddKernel(float *A, float *B, float *C, int n) {
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check index and perform addition
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}