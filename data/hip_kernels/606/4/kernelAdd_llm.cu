#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelAdd(float *dvalues, int numOperations, int firstInd, int nextColInd)
{
    int vi = firstInd + blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure valid index before operations
    if (vi < nextColInd) {
        // Load value into a register to reduce global memory access
        float value = dvalues[vi];
        for (int j = 0; j < numOperations; ++j) {
            // Use register value for operations
            value *= value + value * value;
        }
        // Store the result back to global memory
        dvalues[vi] = value;
    }
}