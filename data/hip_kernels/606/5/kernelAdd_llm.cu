#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelAdd(float *dvalues, int numOperations, int firstInd, int nextColInd)
{
    int vi = firstInd + blockIdx.x * blockDim.x + threadIdx.x;
    
    if (vi < nextColInd) {
        float val = dvalues[vi]; // Load value once to avoid redundant accesses
        for (int j = 0; j < numOperations; ++j) {
            // Use local variable for repeated calculation to reduce global memory access
            // Optimize the operation on the nonzero of the sparse matrix:
            dvalues[vi] /= val + val * val; 
        }
    }
}