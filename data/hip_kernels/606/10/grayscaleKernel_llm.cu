#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void grayscaleKernel(int *ms, int *aux, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    // Use memory coalescing for accessing global memory
    if (i < n) {
        for (int k = 0; k < n - 3; k += 3) {
            // Use shared memory to reduce redundant calculations (Example stub)
            int r = ms[i * n + k];
            int g = ms[i * n + k + 1];
            int b = ms[i * n + k + 2];

            // Precompute constants
            int grayscale = 0.299 * r + 0.5876 * g + 0.114 * b;

            // Write result back to global memory once
            aux[i * n + k] = grayscale;
            aux[i * n + k + 1] = grayscale;
            aux[i * n + k + 2] = grayscale;
        }
    }
}