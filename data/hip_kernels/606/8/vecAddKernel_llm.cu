#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAddKernel(float *A, float *B, float *C, int n) {
    // Use shared memory for input vectors
    extern __shared__ float shared_mem[];
    float *shared_A = shared_mem;
    float *shared_B = shared_mem + blockDim.x;

    int i = threadIdx.x + blockDim.x * blockIdx.x;

    // Load data into shared memory if within bounds
    if (i < n) {
        shared_A[threadIdx.x] = A[i];
        shared_B[threadIdx.x] = B[i];
    }
    // Sync threads in the block to ensure all data is loaded
    __syncthreads();

    // Ensure calculation is done only for valid indices
    if (i < n) {
        C[i] = shared_A[threadIdx.x] + shared_B[threadIdx.x];
    }
}