#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory to optimize memory access
__global__ void addKernel(int * dev_a, int * dev_b, int * dev_c)
{
    int i = threadIdx.x;
    // Check if the thread index is within bounds
    if (i < blockDim.x)
    {
        dev_c[i] = dev_a[i] + dev_b[i];
    }
}