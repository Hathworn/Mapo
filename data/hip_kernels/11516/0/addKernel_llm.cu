#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *dev_a, int* x)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x; // Allow for grid stride
    if (i < *x) // Bounds check to avoid out-of-bound access
    {
        dev_a[i] = (dev_a[i] < *x) ? 0 : 1; // Use ternary operator for simplicity
    }
}