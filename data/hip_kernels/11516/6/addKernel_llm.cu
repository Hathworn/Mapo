#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int * dev_a, int* x)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x; // Support all threads in all blocks
    if (i < *x) // Ensure index is within range
    {
        dev_a[i] = (dev_a[i] < *x) ? 0 : 1; // Use ternary operator for branch optimization
    }
}