#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *dev_a, int *dev_b, int *dev_size)
{
    // Use blockIdx.x and blockDim.x to handle larger data by assigning unique index to each thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int size = *dev_size; // Cache dev_size in shared memory
    if (i < size) {
        int temp = 0; // Use a temporary variable to accumulate sum
        for (int j = 0; j < size; j++) {
            int p = size * i + j;
            temp += dev_a[p];
            //printf("%d %d\n", i, p);
        }
        dev_b[i] = temp; // Assign accumulated value to dev_b
    }
}