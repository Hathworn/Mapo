#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, int *a, int *b)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform addition if index is within bounds (assumes N is the size of the arrays)
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}