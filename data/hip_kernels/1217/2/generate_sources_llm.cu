#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generate_sources(hiprandState *state, int n, uint32_t *verts)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Use a conditional check to avoid out-of-bounds access
    if (tid < n) {
        hiprandState local_state = state[tid];
        for (int id = tid; id < n; id += stride) {
            verts[id] = hiprand(&local_state);
        }
        state[tid] = local_state;
    }
}