#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generate_destinations(hiprandState *state, int n, const uint32_t *sources, uint32_t *destinations)
{
    int first  = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    if (first < n) { // Ensure the first index is within bounds
        hiprandState local_state = state[first];
        for (int id = first; id < n; id += stride) {
            destinations[id] = sources[hiprand(&local_state) % n];
        }
        state[first] = local_state;
    }
}