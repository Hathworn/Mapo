#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_diff(float* a, const float* x, const float* y, const float c, int size) {
    // Calculate a linear index based on block and thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Process only the elements within the valid range
    while (i < size) {
        a[i] += c * (x[i] - y[i]);
        // Advance to the next element in stride of grid size
        i += blockDim.x * gridDim.x;
    }
}