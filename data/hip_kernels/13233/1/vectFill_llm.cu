#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectFill(int *data1, int *data2, int *restult, unsigned long sizeOfArray) {
    unsigned long idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned long stride = blockDim.x * gridDim.x;

    // Use stride loop to optimize memory access and improve parallelism
    for (unsigned long i = idx; i < sizeOfArray; i += stride) {
        restult[i] = data1[i] + data2[i];
    }
}