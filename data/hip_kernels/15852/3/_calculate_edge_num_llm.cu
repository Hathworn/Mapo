#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _calculate_edge_num(long* edge_num, const long* edge_num_sum, const long* edge_idx_sort, const int b, const int n, const int orig_p_num, const long p_num) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= b * n) return;

    const int c_b = index / n;
    const int c_n = index % n;

    long* c_edge_num = &edge_num[c_b * n * orig_p_num + c_n * orig_p_num];
    const long c_edge_num_sum = edge_num_sum[c_b * n + c_n];
    const long* c_edge_idx_sort = &edge_idx_sort[c_b * n * orig_p_num + c_n * orig_p_num];

    if (c_edge_num_sum == p_num) return;

    long diff = p_num - c_edge_num_sum;
    if (diff > 0) {
        // Directly adjust the first sorted edge
        c_edge_num[c_edge_idx_sort[0]] += diff;
    } else {
        // Handle excess edges efficiently
        int id = 0;
        long excess = -diff;
        while (excess > 0) {
            long edge_idx = c_edge_idx_sort[id];
            long decrement = min(excess, c_edge_num[edge_idx] - 1);
            c_edge_num[edge_idx] -= decrement;
            excess -= decrement;
            id++;
        }
    }
}