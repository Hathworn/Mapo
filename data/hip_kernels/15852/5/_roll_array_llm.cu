#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _roll_array(const float* __restrict__ array, const long* __restrict__ step, float* __restrict__ new_array, const int b, const int n, const int d) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= b * n * d)
        return;

    // Precompute constants and modulos
    const int nd = n * d;
    const int c_b = index / nd;
    const int c_n = (index % nd) / d;
    const int c_d = index % d;
    
    const int c_step = static_cast<int>(step[c_b]);
    const int new_n = (c_n + c_step + n) % n;
    const int position = new_n * d + c_d;
    
    // Directly assign the computed position
    new_array[c_b * nd + position] = array[index];
}