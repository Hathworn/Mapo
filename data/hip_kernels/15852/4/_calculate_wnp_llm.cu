#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _calculate_wnp( const long* edge_num, const long* edge_start_idx, float* weight, long* ind, const int b, const int n, const int orig_p_num, const int p_num ) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index >= b * n * orig_p_num)
    return;

  // Pre-compute indices to reduce redundant calculations
  const int c_b = index / (n * orig_p_num);
  const int c_n = (index % (n * orig_p_num)) / orig_p_num;  // Simplified calculation
  const int c_edge_idx = index % orig_p_num;
  
  const long c_edge_num = edge_num[index];
  const int c_start_idx = int(edge_start_idx[index]);
  
  // Pointer calculations outside the loop to improve performance
  float* c_weight = &weight[c_b * n * p_num + c_n * p_num + c_start_idx];
  long* c_ind = &ind[c_b * n * p_num * 2 + c_n * p_num * 2 + c_start_idx * 2];

  // Using temporary variables for improved memory access patterns
  for (long i = 0; i < c_edge_num; i++) {
    float edge_ratio = float(i) / float(c_edge_num);
    c_weight[i] = edge_ratio;
    
    long ind_offset = i * 2;
    c_ind[ind_offset] = long(c_edge_idx);
    c_ind[ind_offset + 1] = long((c_edge_idx + 1) % orig_p_num);
  }
}