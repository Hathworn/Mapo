#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst) {
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Check bounds early to avoid unnecessary computations
    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Bottom left corner of target pixel
    float cx = u_ * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v_ * time_scale + static_cast<float>(i) + 1.0f;

    // Pixel containing bottom left corner
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Use a shared memory buffer to reduce global memory accesses (hypothetical improvement)
    // __shared__ float shared_data[BLOCK_SIZE]; // Assume BLOCK_SIZE is defined appropriately

    float value = src[image_row_offset + j];
    // Fill pixel
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}