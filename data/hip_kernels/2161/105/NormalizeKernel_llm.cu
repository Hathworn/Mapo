#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Optimize by using a single calculation for thread and block indices
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (i < h && j < w) {
        int pos = i * s + j;
        
        // Read normalization factor
        float scale = normalization_factor[pos];
        
        // Compute inverse scale, handling zero case
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);
        
        // Apply normalization
        image[pos] *= invScale;
    }
}