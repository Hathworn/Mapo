#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use a templated kernel to support different vector types and enhance efficiency
template <typename T>
__global__ void scaleVector(T *d_res, const T *d_src, T scale, const int len)
{
    // Calculate global position
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure thread is within bounds
    if (pos < len)
    {
        // Perform scaling operation
        d_res[pos] = d_src[pos] * scale;
    }
}