#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // pixel containing bottom left corner
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    // target pixel integer coords
    int tx = (int) px;
    int ty = (int) py;

    float value = src[image_row_offset + j];
    float weight;

    // Calculate weights and update in a single loop
    float weights[4] = { dx * dy, (1.0f - dx) * dy, (1.0f - dx) * (1.0f - dy), dx * (1.0f - dy) };
    int tx_offsets[4] = { 0, -1, -1, 0 };
    int ty_offsets[4] = { 0, 0, -1, -1 };

    for (int idx = 0; idx < 4; ++idx) {
        int new_tx = tx + tx_offsets[idx];
        int new_ty = ty + ty_offsets[idx];

        if (!(new_tx >= w || new_tx < 0 || new_ty >= h || new_ty < 0)) {
            weight = weights[idx];
            _atomicAdd(dst + new_ty * image_stride + new_tx, value * weight);
            _atomicAdd(normalization_factor + new_ty * image_stride + new_tx, weight);
        }
    }
}