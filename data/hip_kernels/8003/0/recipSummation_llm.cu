#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void recipSummation(double* data, double* recip, int len)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (i < len) {
        const int j = 2 * i;
        data[j]     *= recip[i];
        data[j + 1] *= recip[i];
    }
}