#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello() 
{
    // Calculate global thread ID
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure each thread prints a unique message
    printf("Hello world! I'm thread %d in block %d\n", globalThreadId, blockIdx.x);
}