#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SharedMem2Registers(float * outFloat, int iSize)
{
    /* Amount of shared memory is determined by host call */
    extern __shared__ float s_memoryA[];
    
    /* Generate global index */
    int iID = blockDim.x * blockIdx.x + threadIdx.x;
    
    /* Get the number of available threads */
    int iNumThreads = blockDim.x * gridDim.x;
    
    /* Calculate number of elements */
    int iNumElements = iSize / sizeof(float);
    
    /* Read global memory (coalesce) to shared memory */
    for(int i = iID; i < iNumElements; i += iNumThreads) {
        s_memoryA[i] = outFloat[i]; // Improved reading from global to shared memory
    }
    
    /* Sync all threads to ensure shared memory is fully populated */
    __syncthreads(); // Ensure all threads reach this point before proceeding
    
    /* Variable in register; Read from shared memory to register */
    float r_var = 0.0f;
    if (iID < iNumElements) {
        r_var = s_memoryA[iID];
    }
    
    /* Conditionally assign register var, so it won't get optimized */
    if(iID == 0) outFloat[0] = r_var;
}