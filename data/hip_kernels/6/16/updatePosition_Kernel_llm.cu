#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updatePosition_Kernel(int numElements, float4* bodyPos, float3* bodySpeed)
{
    int elementId = blockIdx.x * blockDim.x + threadIdx.x;

    if (elementId < numElements) {
        float4 elementPosMass = bodyPos[elementId]; // Load position and mass
        float3 elementSpeed = bodySpeed[elementId]; // Load speed

        // Update position using speed and TIMESTEP
        elementPosMass.x += elementSpeed.x * TIMESTEP;
        elementPosMass.y += elementSpeed.y * TIMESTEP;
        elementPosMass.z += elementSpeed.z * TIMESTEP;

        bodyPos[elementId] = elementPosMass; // Store updated position
    }
}