#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SharedMem2globalMem(float *d_memoryA, int iSize)
{
    /* Amount of shared memory is determined by host call */
    extern __shared__ float s_memoryA[];

    /* Generate global index */
    int iID = blockDim.x * blockIdx.x + threadIdx.x;

    /* Calculate number of elements */
    int iNumElements = iSize / sizeof(float);

    /* Utilize optimized stride access pattern */
    for (int i = iID; i < iNumElements; i += blockDim.x * gridDim.x)
    {
        d_memoryA[i] = s_memoryA[i];
    }
}