#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Registers2SharedMem(float * outFloat, int iSize)
{
    extern __shared__ float s_memoryA[];

    // Variable now initialized to zero for safety
    float r_var = 0.0f;

    // Utilize variables for block and grid size to reduce repetitive computation
    int blockSize = blockDim.x;
    int gridSize = gridDim.x;
    int iID = blockSize * blockIdx.x + threadIdx.x;
    int iNumThreads = blockSize * gridSize;
    int iNumElements = iSize / sizeof(float);

    // Improved memory access pattern (increased stride, reducing bank conflicts)
    for(int i = iID; i < iNumElements; i += iNumThreads) {
        s_memoryA[i] = r_var;
    }

    // Use threads much better by completely separating any control flow divergence
    if(iID == 0) {
        outFloat[0] = r_var;
    }
}