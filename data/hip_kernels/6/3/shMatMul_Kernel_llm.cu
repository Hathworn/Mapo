#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shMatMul_Kernel(int matrixSize, float* matrixA, float* matrixB, float* matrixC)
{
    extern __shared__ float sh_Mem[];
    int tilewidth = blockDim.x;
    float *sh_MatrixA = &(sh_Mem[0]);
    float *sh_MatrixB = &(sh_Mem[tilewidth * tilewidth]);

    int elementIdx = blockIdx.x * blockDim.x + threadIdx.x; // Col
    int elementIdy = blockIdx.y * blockDim.y + threadIdx.y; // Row

    int elementId = elementIdy * matrixSize + elementIdx;
    float CValue = 0;
    
    if (elementIdx < matrixSize && elementIdy < matrixSize) {
        for (int m = 0; m < (matrixSize + tilewidth - 1) / tilewidth; ++m) {
            // Load inputs into shared memory safely with boundary checks
            if (threadIdx.x + m * tilewidth < matrixSize)
                sh_MatrixA[tilewidth * threadIdx.y + threadIdx.x] = matrixA[elementIdy * matrixSize + (m * tilewidth + threadIdx.x)];
            else
                sh_MatrixA[tilewidth * threadIdx.y + threadIdx.x] = 0.0f;
                
            if (threadIdx.y + m * tilewidth < matrixSize)
                sh_MatrixB[tilewidth * threadIdx.y + threadIdx.x] = matrixB[elementIdx + (m * tilewidth + threadIdx.y) * matrixSize];
            else
                sh_MatrixB[tilewidth * threadIdx.y + threadIdx.x] = 0.0f;
                
            __syncthreads();

            // Compute matrix multiplication for the sub-matrix
            for (int k = 0; k < tilewidth; ++k)
                CValue += sh_MatrixA[tilewidth * threadIdx.y + k] * sh_MatrixB[tilewidth * k + threadIdx.x];
                
            __syncthreads();
        }

        // Store output value back to global memory
        matrixC[elementId] = CValue;
    }
}