#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/**************************************************************************************************
*
*       Computer Engineering Group, Heidelberg University - GPU Computing Exercise 06
*
*                 Gruppe : gpucomp02
*
*                   File : main.cu
*
*                Purpose : Reduction
*
**************************************************************************************************/


const static int DEFAULT_MATRIX_SIZE = 1024;
const static int DEFAULT_BLOCK_DIM   = 128;

//
// Function Prototypes
//
void printHelp(char *);

//
// CPU
//
__global__ void reduction_KernelOptimized(int numElements, float* dataIn, float* dataOut)
{
    extern __shared__ float sPartArray[];

    const int tid = threadIdx.x;
    unsigned int elementId = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (elementId < numElements) {
        sPartArray[tid] = dataIn[elementId];
    } else { 
        sPartArray[tid] = 0.0f; // Avoid accessing out of bounds
    }
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sPartArray[tid] += sPartArray[tid + s];
        }
        __syncthreads();
    }

    // Write result from block to global memory
    if (tid == 0) {
        dataOut[blockIdx.x] = sPartArray[0];
    }
}