#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bankConflictsRead(float *outFloat, int iStride, unsigned long long *ullTime)
{
    /* Static size of shared memory */
    __shared__ float s_memoryA[2048]; // Use power of 2 size for better alignment

    /* Variable in register */
    float r_var;

    /* Start measure clock cycles */
    unsigned long long startTime = clock64();

    /* Optimize memory access pattern */
    int index = threadIdx.x * iStride;
    if (index < 2048) {
        /* Access data from shared memory to register */
        r_var = s_memoryA[index];
    }

    /* End measure clock cycles */
    unsigned long long endTime = clock64();
    ullTime[threadIdx.x] = endTime - startTime; // Use per-thread timing

    /* Conditionally assign register var to avoid optimization */
    if (threadIdx.x == 0) outFloat[0] = r_var;
}