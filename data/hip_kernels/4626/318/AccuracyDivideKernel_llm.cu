#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Retrieve the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Execute division only if the thread index is zero
    if (idx == 0) {
        *accuracy /= N;
    }
}