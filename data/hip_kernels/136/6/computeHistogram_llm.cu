#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeHistogram(unsigned int *buffer, int size, unsigned int *histo)
{
    __shared__ unsigned int temp[1024];

    // Initialize shared memory
    temp[threadIdx.x] = 0;
    temp[threadIdx.x + 256] = 0;
    temp[threadIdx.x + 512] = 0;
    temp[threadIdx.x + 768] = 0;
    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;
    // Main loop to accumulate values in shared memory
    while (i < size)
    {
        atomicAdd(&temp[buffer[i] & 1023], 1); // Index wrap to reduce bank conflicts
        i += offset;
    }
    __syncthreads();

    // Accumulate shared memory results into global histogram
    atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
    atomicAdd(&(histo[threadIdx.x + 256]), temp[threadIdx.x + 256]);
    atomicAdd(&(histo[threadIdx.x + 512]), temp[threadIdx.x + 512]);
    atomicAdd(&(histo[threadIdx.x + 768]), temp[threadIdx.x + 768]);
}