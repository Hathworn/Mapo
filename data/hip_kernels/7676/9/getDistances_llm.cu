#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// helper for CUDA error handling
__global__ void getDistances(const double* trainingWeights, const double* testWeights, double* distances, std::size_t trainImageNum, std::size_t testImageNum, std::size_t componentNum)
{
    std::size_t row = blockIdx.x;
    std::size_t col = blockIdx.y * blockDim.x + threadIdx.x;

    // Ensure threads are within bounds
    if (col < testImageNum && row < trainImageNum)
    {
        double sum = 0.0;
        // Use registers for faster computations
        #pragma unroll
        for (std::size_t i = 0; i < componentNum; ++i)
        {
            sum += fabs(trainingWeights[i * trainImageNum + row] - testWeights[i * testImageNum + col]);
        }
        distances[row * testImageNum + col] = sum;
    }
}