#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void getSufficientComponentNum(const double* eigenvalues, std::size_t* componentNum, std::size_t eigenRows, double epsilon)
{
    // Use shared memory to accumulate variance
    extern __shared__ double sharedVariance[];
    unsigned int tid = threadIdx.x;
    double localVariance = 0;

    // Each thread calculates a partial sum
    for (std::size_t i = tid; i < eigenRows; i += blockDim.x) {
        localVariance += eigenvalues[i];
    }
    
    // Sum up partial variances using shared memory
    sharedVariance[tid] = localVariance;
    __syncthreads();

    // Reduce partial sums to calculate total variance
    if (tid == 0) {
        double totalVariance = 0;
        for (int i = 0; i < blockDim.x; ++i) {
            totalVariance += sharedVariance[i];
        }
        totalVariance *= eigenRows;

        // Calculate subVariance and explanatoryScore iteratively
        double subVariance = eigenRows * eigenvalues[eigenRows - 1];
        double explanatoryScore = subVariance / totalVariance;
        for (std::size_t compNum = 1; compNum < eigenRows && explanatoryScore <= epsilon; ++compNum) {
            subVariance += eigenRows * eigenvalues[eigenRows - compNum - 1];
            explanatoryScore = subVariance / totalVariance;
            *componentNum = compNum + 1; // Update componentNum
        }
    }
}