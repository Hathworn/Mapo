#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Helper for CUDA error handling
__global__ void restoreEigenvectors(const double* meanSubtractedImages, const double* reducedEigenvectors, double* restoredEigenvectors, std::size_t imageNum, std::size_t pixelNum, std::size_t componentNum) {
    std::size_t col = blockIdx.x * blockDim.x + threadIdx.x; // Optimize col calculation by using blockIdx.x
    std::size_t row = blockIdx.y * blockDim.y + threadIdx.y; // Optimize row calculation by using blockIdx.y

    // Boundary check
    if (col >= pixelNum || row >= componentNum) return;
    
    // Initialize value for each thread
    double sum = 0.0;
    
    // Perform calculation
    for (std::size_t i = 0; i < imageNum; ++i) {
        sum += reducedEigenvectors[(imageNum - row - 1) * imageNum + i] * meanSubtractedImages[i * pixelNum + col];
    }
    
    // Write the result
    restoredEigenvectors[row * pixelNum + col] = sum;
}