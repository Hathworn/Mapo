#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// helper for CUDA error handling
__global__ void subtractMean(double* __restrict__ images, const double* __restrict__ meanImage, std::size_t imageNum, std::size_t pixelNum)
{
    std::size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (col >= pixelNum)
    {
        return;
    }
    
    // Use shared memory for meanImage if needed across threads
    // Optimize memory access pattern for coalescing
    for (std::size_t row = 0; row < imageNum; ++row)
    {
        std::size_t index = row * pixelNum + col;
        double newValue = images[index] - meanImage[col];
        
        // Directly assign the max operation result
        images[index] = newValue < 0.0 ? 0.0 : newValue;
    }
}