#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// helper for CUDA error handling
__global__ void getMeanImage(const double* images, double* meanImage, std::size_t imageNum, std::size_t pixelNum)
{
    std::size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (col >= pixelNum)
    {
        return;
    }

    // Utilize shared memory to reduce global memory access
    extern __shared__ double sharedImages[];

    double sum = 0.0;
    for (std::size_t row = 0; row < imageNum; ++row)
    {
        // Load image data into shared memory
        sharedImages[threadIdx.x] = images[row * pixelNum + col];
        __syncthreads();

        // Accumulate the sum of pixel values
        sum += sharedImages[threadIdx.x];
        __syncthreads();
    }

    // Compute the mean for each pixel
    meanImage[col] = sum / imageNum;
}