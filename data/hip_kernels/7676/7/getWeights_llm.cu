#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function to compute weights
__global__ void getWeights(const double* restoredEigenvectors, const double* meanSubtractedImages, double* weights, std::size_t imageNum, std::size_t pixelNum, std::size_t componentNum)
{
    std::size_t col = blockIdx.x * blockDim.x + threadIdx.x; // Optimize thread index calculation
    std::size_t row = blockIdx.y * blockDim.y + threadIdx.y; // Optimize block index calculation

    if (col >= imageNum || row >= componentNum)
    {
        return;
    }

    double sum = 0.0; // Use a local variable to accumulate results for better memory access
    for (std::size_t i = 0; i < pixelNum; ++i)
    {
        sum += restoredEigenvectors[row * pixelNum + i] * meanSubtractedImages[col * pixelNum + i];
    }

    weights[row * imageNum + col] = sum; // Write the accumulated result to global memory
}