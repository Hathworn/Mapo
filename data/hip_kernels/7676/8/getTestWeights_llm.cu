#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getTestWeights(const double* restoredEigenvectors, const double* meanImage, const double* testImages, double* testWeights, std::size_t testImageNum, std::size_t pixelNum, std::size_t componentNum)
{
    std::size_t row = blockIdx.x;
    std::size_t col = blockIdx.y * blockDim.x + threadIdx.x;

    // Check bounds
    if (col >= testImageNum || row >= componentNum) {
        return;
    }

    double weight = 0.0;
    for (std::size_t i = 0; i < pixelNum; ++i) {
        double testImagePixelValue = testImages[col * pixelNum + i] - meanImage[i];
        // Directly zero negative values
        if (testImagePixelValue < 0.0) {
            testImagePixelValue = 0.0;
        }
        weight += restoredEigenvectors[row * pixelNum + i] * testImagePixelValue;
    }

    // Write final weight once per row-column pair
    testWeights[row * testImageNum + col] = weight;
}