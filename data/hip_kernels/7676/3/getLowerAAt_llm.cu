#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// helper for CUDA error handling
__global__ void getLowerAAt( const double* A, double* S, std::size_t imageNum, std::size_t pixelNum )
{
    // Calculate unique row and col for each thread
    std::size_t row = blockIdx.x;
    std::size_t col = blockIdx.y * blockDim.x + threadIdx.x;

    // Early return if thread does not contribute
    if (row >= imageNum || col >= imageNum)
    {
        return;
    }

    // Use shared memory to reduce global memory accesses
    extern __shared__ double sharedA[];
    double sum = 0.0;

    for (std::size_t i = 0; i < pixelNum; ++i)
    {
        // Load current row data into shared memory
        sharedA[threadIdx.x] = A[row * pixelNum + i];
        __syncthreads();  // Make sure the data is loaded

        // Calculate partial sum
        sum += sharedA[threadIdx.x] * A[col * pixelNum + i];
        __syncthreads();  // Sync threads before next iteration
    }

    // Write sum to global memory
    S[row * imageNum + col] = sum;
}