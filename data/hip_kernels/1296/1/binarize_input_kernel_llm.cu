#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;

    float mean = 0.0f;
    // Loop unrolling to reduce loop overhead and improve data reuse
    for (int i = 0; i < n; i += 4) {
        if (i < n) mean += fabsf(input[i * size + s]);
        if (i + 1 < n) mean += fabsf(input[(i + 1) * size + s]);
        if (i + 2 < n) mean += fabsf(input[(i + 2) * size + s]);
        if (i + 3 < n) mean += fabsf(input[(i + 3) * size + s]);
    }
    mean /= n;

    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}