#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    if (i < n) {
        // Use conditional operator to simplify calculation
        float b_val = b ? b[i] : 0.0f;
        c[i] = s[i] * a[i] + (1.0f - s[i]) * b_val;
    }
}