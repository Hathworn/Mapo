#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    extern __shared__ float part[]; // Use dynamic shared memory
    int i, b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Unroll loop for performance
    for(b = 0; b < batch; ++b){
        for(i = p; i < size; i += BLOCK){
            int index = i + size * (filter + n * b);
            sum += delta[index] * x_norm[index];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Efficient reduction within a block
    if (BLOCK >= 512) { if (p < 256) part[p] += part[p + 256]; __syncthreads(); }
    if (BLOCK >= 256) { if (p < 128) part[p] += part[p + 128]; __syncthreads(); }
    if (BLOCK >= 128) { if (p <  64) part[p] += part[p +  64]; __syncthreads(); }
    if (p < 32) { // Warp-synchronous on a single warp
        if (BLOCK >=  64) part[p] += part[p + 32];
        if (BLOCK >=  32) part[p] += part[p + 16];
        if (BLOCK >=  16) part[p] += part[p +  8];
        if (BLOCK >=   8) part[p] += part[p +  4];
        if (BLOCK >=   4) part[p] += part[p +  2];
        if (BLOCK >=   2) part[p] += part[p +  1];
    }

    if (p == 0) scale_updates[filter] += part[0]; // Store the result
}