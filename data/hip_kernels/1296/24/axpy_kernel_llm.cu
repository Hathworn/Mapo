#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX,  float *Y, int OFFY, int INCY)
{
    // Use blockIdx.x directly for the index calculation
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    // Loop over the elements this thread is responsible for
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        Y[OFFY + idx * INCY] += ALPHA * X[OFFX + idx * INCX];
    }
}