#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize calculation of 'i' to use 1D grid
    int stride = blockDim.x * gridDim.x; // Calculate stride for coalesced access

    for (; i < N; i += stride) { // Process elements in a loop with stride
        X[i * INCX] *= ALPHA; // Optimize memory access pattern
    }
}