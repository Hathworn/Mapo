#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify index calculation
    int gridStride = gridDim.x * blockDim.x;        // Calculate total grid stride

    for (; i < N; i += gridStride) {                // Use grid-stride loop for balance
        Y[i * INCY] *= X[i * INCX];
    }
}