#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    if (index < N) {  // Check if index is within bounds
        int f = (index / spatial) % filters;
        float var = variance[f] + 0.00001f;  // Precompute for efficiency
        x[index] = (x[index] - mean[f]) / sqrtf(var);
    }
}