#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel by using shared memory
__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    extern __shared__ float shared_biases[];  // Use shared memory for biases
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if (threadIdx.x == 0) {
        shared_biases[filter] = biases[filter];  // Load bias into shared memory
    }
    __syncthreads();  // Ensure all threads have loaded the shared bias

    if (offset < size) {
        output[(batch*n+filter)*size + offset] *= shared_biases[filter];  // Use shared bias
    }
}