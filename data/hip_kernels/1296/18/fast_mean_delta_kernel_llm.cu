#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    int offset = filter * spatial;
    local[id] = 0;

    #pragma unroll // Unroll the loop for optimization
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + offset + i;
            local[id] += delta[index];
        }
    }

    __syncthreads();

    // Use parallel reduction to sum up local array
    for (int stride = threads / 2; stride > 0; stride /= 2) {
        if (id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    if (id == 0) {
        mean_delta[filter] = local[0] * (-1.f / sqrtf(variance[filter] + .00001f));
    }
}