#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Early exit if index exceeds the bounds
    if(i >= N) return;

    int in_index = i;

    // Calculate indices using modulo and division
    int b = i / (w * h * c);
    int in_c = (i / (w * h)) % c;
    int in_h = (i / w) % h;
    int in_w = i % w;

    // Optimize calculations by reusing divisions and avoiding redundant operations
    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;

    // Calculate output index directly using optimized w2, h2, and c2
    int out_index = b * out_c * h * w * (stride * stride) + c2 * h * w * (stride * stride) + h2 * w * stride + w2;
    
    if(forward) 
        out[out_index] = x[in_index];
    else 
        out[in_index] = x[out_index];
}