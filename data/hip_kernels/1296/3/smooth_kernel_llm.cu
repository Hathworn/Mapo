#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    // Calculate global thread index
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Indices for width, height, channels, and batch
    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    // Pre-compute offsets
    int w_offset = -(size / 2);
    int h_offset = -(size / 2);
    int out_index = j + w * (i + h * (k + c * b));

    // Update delta based on valid neighbor pixels
    for (int l = 0; l < size; ++l) {
        int cur_h = h_offset + i + l;
        if (cur_h >= 0 && cur_h < h) {  // Check height bounds
            for (int m = 0; m < size; ++m) {
                int cur_w = w_offset + j + m;
                if (cur_w >= 0 && cur_w < w) {  // Check width bounds
                    int index = cur_w + w * (cur_h + h * (k + b * c));
                    delta[out_index] += rate * (x[index] - x[out_index]);
                }
            }
        }
    }
}