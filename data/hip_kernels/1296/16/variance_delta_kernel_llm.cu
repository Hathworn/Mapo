#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Initialize variance_delta at the start
    float temp_variance_delta = 0.0f;

    // Calculate variance_delta
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            temp_variance_delta += delta[index] * (x[index] - mean[i]);
        }
    }

    // Update variance_delta with computed value and scaling factor
    variance_delta[i] = temp_variance_delta * -.5f * powf(variance[i] + .00001f, -1.5f);
}