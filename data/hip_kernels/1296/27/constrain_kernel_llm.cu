#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use GPU's built-in functions to calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockDim.x * gridDim.x * blockIdx.y;
    if(i < N) {
        // Apply constraint using fminf and fmaxf
        float val = X[i * INCX];
        X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, val));
    }
}