#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate 1D index within grid
    int stride = gridDim.x * blockDim.x;            // Determine stride for grid-wide loop
    for (; i < N; i += stride) {                    // Loop over elements in a grid-stride
        Y[i * INCY] = powf(X[i * INCX], ALPHA);     // Use fast math function powf for float
    }
}