#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_softmax_layer_kernel(int n, int batch, float *input, float temp, float *output)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if(b >= batch) return;

    float sum = 0;
    float largest = -INFINITY;

    // Use shared memory to reduce global memory access
    extern __shared__ float shared_mem[];
    float *shared_input = shared_mem;
    float *shared_output = shared_input + n * blockDim.x;

    // Load inputs to shared memory
    for(int i = threadIdx.x; i < n; i += blockDim.x) {
        shared_input[i] = input[i + b * n];
    }
    __syncthreads();

    // Find the largest element
    for(int i = 0; i < n; ++i) {
        largest = max(largest, shared_input[i]);
    }
    __syncthreads();

    // Compute the exponential values and the sum
    for(int i = 0; i < n; ++i) {
        shared_output[i] = exp((shared_input[i] - largest) / temp);
        sum += shared_output[i];
    }
    __syncthreads();

    // Normalize the output
    for(int i = 0; i < n; ++i) {
        output[i + b * n] = shared_output[i] / sum;
    }
}