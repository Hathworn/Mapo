#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes) {
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int c = in_c;
    int area = (size - 1) / stride;

    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Compute the index positions in the flattened input array
    int index = id;
    int j = index % in_w;
    index /= in_w;
    int i = index % in_h;
    index /= in_h;
    int k = index % in_c;
    index /= in_c;
    int b = index;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    float d = 0;

    // Using one loop for better warp execution
    for (int m = -area; m <= area; ++m) {
        for (int l = -area; l <= area; ++l) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;

            // Calculate output index
            int out_index = out_w + w * (out_h + h * (k + c * b));
            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h && indexes[out_index] == (i * in_w + j + in_h * in_c * b + k * in_h * in_w)) {
                d += delta[out_index];
            }
        }
    }
    prev_delta[i * in_w + j + in_h * in_c * b + k * in_h * in_w] += d;
}