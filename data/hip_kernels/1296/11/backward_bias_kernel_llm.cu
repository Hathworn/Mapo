#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x; 
    int p = threadIdx.x;
    float sum = 0;

    // Improved to unroll loop over 'i' for better performance
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; i += BLOCK) {
            int index = p + i + size * (filter + n * b);
            sum += (p + i < size) ? delta[index] : 0;
        }
    }

    part[p] = sum;
    __syncthreads();

    // Use reduction within shared memory to compute the final sum, instead of single thread addition
    if (p < 32) {
        for (int offset = BLOCK / 2; offset > 0; offset /= 2) {
            if (p < offset) {
                part[p] += part[p + offset];
            }
            __syncthreads();
        }
    }
    
    if (p == 0) {
        atomicAdd(&bias_updates[filter], part[0]); // Use atomic add to safely update global memory
    }
}