#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use only blockIdx.x for 1D grid
    if (i >= filters) return;

    float sum = 0.0f; // Use a local variable for sum
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += x[index]; // Accumulate sum
        }
    }
    mean[i] = sum * scale; // Multiply by scale after loop
}