#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    // Initialize local array for each thread
    local[id] = 0;
    
    // Loop unrolling for spatial dimension
    int index_base = filter * spatial;
    for (int j = 0; j < batch; ++j) {
        #pragma unroll
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + index_base + i;
            if (i < spatial) {
                local[id] += x[index];
            }
        }
    }

    __syncthreads();

    // Efficient reduction for local sums
    if (id < 32) {
        for (int offset = threads / 2; offset > 0; offset /= 2) {
            local[id] += __shfl_down_sync(0xffffffff, local[id], offset);
        }
    }

    if (id == 0) {
        mean[filter] = local[0] / (spatial * batch);
    }
}