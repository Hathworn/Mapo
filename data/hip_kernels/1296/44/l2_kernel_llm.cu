#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Using blockDim.x * gridDim.x to compute unique index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Handling boundary cases where i might exceed n
    if (i < n) {
        float diff = truth[i] - pred[i];
        error[i] = diff * diff; // Calculate squared error
        delta[i] = diff;        // Store the difference
    }
}