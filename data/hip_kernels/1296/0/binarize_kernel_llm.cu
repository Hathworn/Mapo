#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure within bounds
    if (i < n) 
    {
        // Perform binary operation
        binary[i] = (x[i] >= 0) ? 1 : -1;
    }
}