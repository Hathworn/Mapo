#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    // Initialize local memory with parallel threads
    local[id] = 0;
    __syncthreads(); // Ensures full initialization before use

    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) { // Optimize loop iteration by starting at 'id' step
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += powf((x[index] - mean[filter]), 2);
        }
    }

    __syncthreads();

    // Use reduction pattern for efficiency
    for (int stride = threads / 2; stride > 0; stride /= 2) {
        if (id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    if (id == 0) {
        variance[filter] = local[0] / (spatial * batch - 1);
    }
}