#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    extern __shared__ float local[];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    // Each thread computes its contribution to shared memory
    float sum = 0.0f;
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += blockDim.x) {
            int index = j * spatial * filters + filter * spatial + i;
            if (i < spatial) {
                sum += delta[index] * (x[index] - mean[filter]);
            }
        }
    }
    local[id] = sum;

    __syncthreads();

    // Use a parallel reduction to compute the block-wide sum
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (id < offset) {
            local[id] += local[id + offset];
        }
        __syncthreads();
    }

    // First thread computes the final result for this block
    if (id == 0) {
        variance_delta[filter] = local[0] * (-0.5f * powf(variance[filter] + 0.00001f, -1.5f));
    }
}