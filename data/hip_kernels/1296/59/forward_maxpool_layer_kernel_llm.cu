#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    int h = (in_h + pad - size)/stride + 1;
    int w = (in_w + pad - size)/stride + 1;
    int c = in_c;

    // Use threadIdx.x to assign unique thread id
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int b = id / (w * h * c);
    int k = (id / (w * h)) % c;
    int i = (id / w) % h;
    int j = id % w;

    int w_offset = -pad/2;
    int h_offset = -pad/2;

    int out_index = j + w * (i + h * (k + c * b));
    float max = -INFINITY;
    int max_i = -1;
    int index, cur_h, cur_w;

    // Unroll loops for better performance
    #pragma unroll
    for (int l = 0; l < size; ++l) {
        #pragma unroll
        for (int m = 0; m < size; ++m) {
            cur_h = h_offset + i * stride + l;
            cur_w = w_offset + j * stride + m;
            index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            bool valid = (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w);
            float val = (valid) ? input[index] : -INFINITY;
            if (val > max) {
                max = val;
                max_i = index;
            }
        }
    }
    
    output[out_index] = max;
    indexes[out_index] = max_i;
}