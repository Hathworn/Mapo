#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    // Check if the thread id is within the valid range
    if(id >= n) return;
    
    int k = id % c;
    int b = id / c;
    int out_index = k + c * b;

    // Initialize output element
    float sum = 0.0f;
    
    // Optimize memory access by using local variable 'sum'
    for(int i = 0; i < w * h; ++i){
        int in_index = i + h * w * (k + b * c);
        sum += input[in_index];
    }
    // Finalize the average for current output element
    output[out_index] = sum / (w * h);
}