#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified block index calculation
    const int stride = blockDim.x * gridDim.x;      // Calculate stride for loop unrolling

    while (i < n) {                                 // Loop over the indices using stride
        float diff = truth[i] - pred[i];
        error[i] = fabsf(diff);                     // Use fast math function for absolute value
        delta[i] = copysignf(1.0f, diff);           // Use fast math function for sign
        i += stride;                                // Move to next index
    }
}