#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Use shared memory for faster access within the block
    extern __shared__ float shared_weights[];
    
    // Load weights into shared memory and calculate mean
    float mean = 0;
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[i] = fabsf(weights[f * size + i]);
        atomicAdd(&mean, shared_weights[i]);
    }
    __syncthreads();

    // Calculate mean average
    mean = mean / size;

    // Binarize weights using precomputed mean
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    }
}