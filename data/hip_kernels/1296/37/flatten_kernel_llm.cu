#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Use shared memory for spatial and layers indices to reduce calculations
    __shared__ int shared_spatial[1024]; // Assume blockDim.x <= 1024
    __shared__ int shared_layers[1024];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    int in_s = i % spatial;
    int in_c = (i / spatial) % layers;
    int b = i / (spatial * layers);

    // Precompute offsets
    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;

    // Conditional assignment using ternary operator
    out[forward ? i2 : i1] = x[forward ? i1 : i2];
}