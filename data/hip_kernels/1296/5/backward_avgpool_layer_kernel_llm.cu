#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int k = id % c; // Calculate channel index
    id /= c;
    int b = id; // Calculate batch index

    int out_index = k + c * b;
    float scale = 1.0f / (w * h); // Precompute division for optimization

    // Consolidate computation and memory access to improve performance
    for (int i = 0; i < w * h; ++i) {
        int in_index = i + w * h * out_index;
        in_delta[in_index] += out_delta[out_index] * scale;
    }
}