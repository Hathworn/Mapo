#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index with 3D block and grid consideration:
    int i = blockIdx.x * blockDim.x + threadIdx.x + 
            blockIdx.y * blockDim.x * gridDim.x + 
            blockIdx.z * blockDim.x * gridDim.x * gridDim.y;
    
    // Efficient boundary check and update operation:
    if(i < N) X[i * INCX] += ALPHA;
}