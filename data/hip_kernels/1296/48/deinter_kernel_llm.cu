#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *out)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Check if thread index is within bounds
    if (i < (NX + NY) * B) {
        int b = i / (NX + NY);
        int j = i % (NX + NY);

        // Perform element-wise operation with boundary check
        if (j < NX && X) {
            X[b * NX + j] += out[i];
        } else if (Y) {
            Y[b * NY + j - NX] += out[i];
        }
    }
}