#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Calculate index, f1, and f2 for the matrix element
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;

    // Combined loops to maximize data access coalescing
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            float val1 = output[i1];
            float val2 = output[i2];
            sum += val1 * val2;
            norm1 += val1 * val1;
            norm2 += val2 * val2;
        }
    }

    // Moved square root calculation outside the loop to reduce redundant calculations
    norm1 = sqrtf(norm1);
    norm2 = sqrtf(norm2);
    float norm = norm1 * norm2;

    // Prevent division by zero if norm is zero
    if (norm == 0.0f) return;

    sum /= norm;

    // Combined loops for delta updates to improve memory access patterns
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            float scaleSumOutput2 = -scale * sum * output[i2] / norm;
            float scaleSumOutput1 = -scale * sum * output[i1] / norm;
            delta[i1] += scaleSumOutput2;
            delta[i2] += scaleSumOutput1;
        }
    }
}