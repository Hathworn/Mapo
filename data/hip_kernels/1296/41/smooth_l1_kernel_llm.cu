#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds
    if (i < n) {
        // Calculate difference and its absolute value
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);

        // Use conditional ternary operator for concise operation
        error[i] = (abs_val < 1) ? diff * diff : 2 * abs_val - 1;
        delta[i] = (abs_val < 1) ? diff : (diff > 0 ? 1 : -1);
    }
}