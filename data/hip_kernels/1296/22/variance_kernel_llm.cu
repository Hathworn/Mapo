#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Pre-calculate constants outside the loop to reduce repeated computation
    float scale = 1.f / (batch * spatial - 1);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i >= filters) return;
    
    float var = 0.0f; // Use a local variable to avoid redundant global memory access
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            var += powf((x[index] - mean[i]), 2);
        }
    }
    
    variance[i] = var * scale; // Assign the final result to the global memory
}