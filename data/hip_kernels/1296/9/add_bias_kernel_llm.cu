#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate the global thread ID
    int stride = blockDim.x * gridDim.x;  // Determine the stride

    // Iterate over elements in increments of stride size
    for (int index = idx; index < n * size * batch; index += stride) {
        int i = index % size;
        int j = (index / size) % n;
        int k = index / (size * n);
        
        output[(k * n + j) * size + i] += biases[j];  // Add bias to the output
    }
}