#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to store variance and mean values per filter
    __shared__ float shared_variance[blockDim.x];
    __shared__ float shared_mean[blockDim.x];

    if (threadIdx.x < filters) {
        shared_variance[threadIdx.x] = variance[threadIdx.x];
        shared_mean[threadIdx.x] = mean[threadIdx.x];
    }
    __syncthreads();

    if (index >= N) return;

    int f = (index / spatial) % filters;

    // Use shared memory value instead of global for improved performance
    float var_f = shared_variance[f];
    float mean_f = shared_mean[f];

    delta[index] = delta[index] * __frsqrt_rn(var_f + .00001f) + variance_delta[f] * 2.f * (x[index] - mean_f) / (spatial * batch) + mean_delta[f] / (spatial * batch);
}