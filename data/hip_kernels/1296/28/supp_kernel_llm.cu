#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use grid-stride loop for better utilization of GPU resources
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        if ((X[i*INCX] * X[i*INCX]) < (ALPHA * ALPHA)) {
            X[i*INCX] = 0;
        }
    }
}