#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Ensure thread index is within array bounds
    if(i < n) {
        float t = truth[i];
        float p = pred[i];
        
        // Pre-calculate common expressions for performance improvement
        float epsilon = 0.0000001;
        float log_p = log(p + epsilon);
        float log_1mp = log(1 - p + epsilon);

        // Compute error and delta values
        error[i] = -t * log_p - (1 - t) * log_1mp;
        delta[i] = t - p;
    }
}