#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation
    int stride = gridDim.x * blockDim.x; // Calculate stride for grid-stride loop
    for (; i < n; i += stride) { // Use grid-stride loop to process more data per block
        if (mask[i] == mask_num) x[i] *= scale; // Multiply if condition is met
    }
}