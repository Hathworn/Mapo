#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
  // Calculate unique index for each thread
  size_t i = blockIdx.x * blockDim.x + threadIdx.x + blockDim.x * gridDim.x * blockIdx.y; 
  if(i >= N) return; // Exit if index exceeds total threads

  // Calculate indices directly to reduce operations
  int out_w = i % (w * stride);
  int out_h = (i / (w * stride)) % (h * stride);
  int out_c = (i / (w * stride * h * stride)) % c;
  int b = i / (w * stride * h * stride * c);

  // Calculate input indices for source matrix
  int in_w = out_w / stride;
  int in_h = out_h / stride;

  int in_index = b * w * h * c + out_c * w * h + in_h * w + in_w; // Simplified in_index computation

  // Conditional operation based on forward variable
  if(forward) {
    out[out_index] += scale * x[in_index]; // Scale and add for forward pass
  } else {
    atomicAdd(x + in_index, scale * out[out_index]); // Use atomicAdd for backward pass
  }
}