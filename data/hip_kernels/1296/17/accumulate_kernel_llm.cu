#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    // Initialize sum to zero
    float totalSum = 0.0f;

    // Perform accumulation
    for (int k = 0; k < n; ++k) {
        totalSum += x[k * groups + i];
    }

    // Write result to global memory
    sum[i] = totalSum;
}