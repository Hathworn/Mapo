#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Check boundary condition and perform calculations
    if (i < n) {
        float dc_val = dc[i];
        float s_val = s[i];
        
        if (da) da[i] += dc_val * s_val;
        if (db) db[i] += dc_val * (1 - s_val);
        ds[i] += dc_val * (a[i] - b[i]);
    }
}