#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *out)
{
    // Calculate a unique global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate the total number of elements
    int total_elements = (NX + NY) * B;
    
    // Ensure thread index is within bounds
    if (i < total_elements) {
        int b = i / (NX + NY); // Determine the batch index
        int j = i % (NX + NY); // Determine the local index within the batch

        // Conditionally load data from X or Y based on index
        if (j < NX) {
            out[i] = X[b * NX + j]; // Load from array X
        } else {
            out[i] = Y[b * NY + (j - NX)]; // Load from array Y
        }
    }
}