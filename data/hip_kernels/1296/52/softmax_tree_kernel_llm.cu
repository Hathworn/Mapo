#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;

    // Use shared memory for faster access
    extern __shared__ float shared_input[];
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        shared_input[i] = input[i * stride];
    }
    __syncthreads();

    for (i = threadIdx.x; i < n; i += blockDim.x) {
        float val = shared_input[i];
        atomicMax(&largest, val); // Determine the largest value
    }
    __syncthreads();

    for (i = threadIdx.x; i < n; i += blockDim.x) {
        float e = expf(shared_input[i] / temp - largest / temp);
        atomicAdd(&sum, e); // Calculate the sum
        shared_input[i] = e;
    }
    __syncthreads();

    for (i = threadIdx.x; i < n; i += blockDim.x) {
        output[i * stride] = shared_input[i] / sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;

    int s = id % spatial;
    id = id / spatial;
    int g = id % groups;
    int b = id / groups;
    int goff = group_offset[g] * spatial;
    int boff = b * stride;

    // Use shared memory of appropriate size
    extern __shared__ float shared_memory[];
    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}