#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate global thread index using 1D grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Loop through data in steps of total threads in grid
    for (; i < n; i += blockDim.x * gridDim.x) {
        c[i] += a[i] * b[i];
    }
}