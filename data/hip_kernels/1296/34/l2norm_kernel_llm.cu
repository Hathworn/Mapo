#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;

    int b = index / spatial;  // Batch index
    int i = index % spatial;  // Spatial index
    float sum = 0;

    // Unrolling filter loop for efficiency
    int f;
    for (f = 0; f < filters; f += 4) {
        int index_base = b * filters * spatial + i;  // Base index for this batch and spatial
        if(f + 0 < filters) {
            int idx0 = index_base + (f + 0) * spatial;
            sum += x[idx0] * x[idx0];
        }
        if(f + 1 < filters) {
            int idx1 = index_base + (f + 1) * spatial;
            sum += x[idx1] * x[idx1];
        }
        if(f + 2 < filters) {
            int idx2 = index_base + (f + 2) * spatial;
            sum += x[idx2] * x[idx2];
        }
        if(f + 3 < filters) {
            int idx3 = index_base + (f + 3) * spatial;
            sum += x[idx3] * x[idx3];
        }
    }

    sum = sqrtf(sum);
    if (sum == 0) sum = 1;

    for (f = 0; f < filters; ++f) {
        int index = b * filters * spatial + f * spatial + i;
        x[index] /= sum;
        dx[index] = (1 - x[index]) / sum;
    }
}