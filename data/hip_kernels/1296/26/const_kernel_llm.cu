#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplify and optimize index calculation
    int stride = blockDim.x * gridDim.x; // Define stride for grid-stride loop

    for (int i = idx; i < N; i += stride) // Use grid-stride loop for better performance
    {
        X[i * INCX] = ALPHA; // Update with ALPHA
    }
}