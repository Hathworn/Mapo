#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;  // Calculate total number of threads processing the array
    
    // Use strided loop to allow multiple blocks to fill the array
    for (; i < N; i += stride) {
        X[i * INCX] = ALPHA;
    }
}