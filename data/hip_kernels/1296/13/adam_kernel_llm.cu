#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation for 1D grid
    if (index >= N) return;

    // Pre-compute common terms outside of index-based computations
    float B1_pow_t = powf(B1, t);
    float B2_pow_t = powf(B2, t);

    // Use these precomputed variables downstream for efficiency
    float mhat = m[index] / (1.f - B1_pow_t);
    float vhat = v[index] / (1.f - B2_pow_t);

    x[index] = x[index] + rate * mhat / (sqrtf(vhat) + eps);
}