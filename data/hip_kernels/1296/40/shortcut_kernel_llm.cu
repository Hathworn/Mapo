#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) return;

    // Calculate indices efficiently
    int b = id / (minw * minh * minc);
    int rest = id % (minw * minh * minc);
    int k = rest / (minw * minh);
    rest = rest % (minw * minh);
    int j = rest / minw;
    int i = rest % minw;

    // Calculate output and add indices
    int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
    int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));

    // Perform computation with simplified access
    out[out_index] = s1 * out[out_index] + s2 * add[add_index];
}