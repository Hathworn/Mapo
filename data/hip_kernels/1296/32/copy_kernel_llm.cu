#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x; // Compute total grid size
    
    for (; i < N; i += gridSize) { // Loop to handle more work per thread
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
    }
}