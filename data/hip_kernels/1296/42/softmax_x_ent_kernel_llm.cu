#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified grid index calculation

    // Vectorized access pattern to maximize memory coalescing and reduce divergence
    for (; i < n; i += blockDim.x * gridDim.x)
    {
        float t = truth[i];
        float p = pred[i];
        error[i] = (t) ? -log(p) : 0;
        delta[i] = t - p;
    }
}