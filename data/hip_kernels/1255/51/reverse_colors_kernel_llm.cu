#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reverse_colors_kernel(int num_rows, int max_color, int *row_colors)
{
    int row_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Using strided loop to handle more work per warp
    for ( ; row_id < num_rows ; row_id += blockDim.x * gridDim.x )
    {
        int color = row_colors[row_id];

        // Conditional check and assignment combined for efficiency
        if (color > 0)
        {
            // Reverse the color value
            row_colors[row_id] = max_color - color + 1;
        }
        else
        {
            row_colors[row_id] = color;
        }
    }
}