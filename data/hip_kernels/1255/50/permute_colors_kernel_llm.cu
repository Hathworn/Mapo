#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void permute_colors_kernel(int num_rows, int *row_colors, int *color_permutation)
{
    int row_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a while loop for simpler iteration control
    while (row_id < num_rows)
    {
        int color = row_colors[row_id];

        // Simplify condition checking with HIP built-in function support for constant memory
        color = __ldg(&color_permutation[color]);

        row_colors[row_id] = color;

        // Advance to the next row_id using grid-stride loop
        row_id += blockDim.x * gridDim.x;
    }
}