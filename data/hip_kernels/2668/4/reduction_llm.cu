#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction(int* input, int* output) {
    __shared__ int tmp[TPB];

    int tid = threadIdx.x;
    int index = tid + blockIdx.x * blockDim.x;
    
    // Load data
    tmp[tid] = input[index];
    __syncthreads();

    // Reduce using a while loop to simplify logic
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            tmp[tid] += tmp[tid + stride];
        }
        __syncthreads();
    }

    // Write result
    if (tid == 0) {
        output[blockIdx.x] = tmp[0];
    }
}