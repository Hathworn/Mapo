#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    // Use shared memory for vector_in to reduce global memory access
    __shared__ double shared_vector[1024]; // Assumes max threads per block is 1024
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    long tid = threadIdx.x;
    
    if (tid < dim_mn) {
        shared_vector[tid] = vector_in[tid];
    }
    
    __syncthreads(); // Ensure shared_vector is fully loaded before proceeding
    
    if (i < dim_mn) {
        double out = 0.0;
        for (long j = 0; j < dim_mn; j++) {
            out += matrix[i * dim_mn + j] * shared_vector[j];
        }
        vector_out[i] = out;
    }
}