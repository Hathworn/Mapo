#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bigstencil(int* in, int* out) {
    // Utilize the shared memory for faster access
    extern __shared__ int shared_in[];
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    shared_in[threadIdx.x] = in[i]; // Load input into shared memory
    __syncthreads(); // Ensure all threads have loaded data into shared memory

    // Perform computation using shared memory
    out[i] = shared_in[threadIdx.x] + 2;
}