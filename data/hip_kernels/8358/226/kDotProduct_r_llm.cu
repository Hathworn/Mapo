#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Loop unrolling for improved memory coalescing
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        sum += a[eidx] * b[eidx];
    }

    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Efficient parallel reduction with unrolling
    if (DP_BLOCKSIZE >= 512) { if (threadIdx.x < 256) { shmem[threadIdx.x] += shmem[threadIdx.x + 256]; } __syncthreads(); }
    if (DP_BLOCKSIZE >= 256) { if (threadIdx.x < 128) { shmem[threadIdx.x] += shmem[threadIdx.x + 128]; } __syncthreads(); }
    if (DP_BLOCKSIZE >= 128) { if (threadIdx.x <  64) { shmem[threadIdx.x] += shmem[threadIdx.x + 64]; } __syncthreads(); }

    // Warp-level reduction with no additional sync needed
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}