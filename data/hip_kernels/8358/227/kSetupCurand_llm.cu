#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate unique thread id
    const uint tidx = blockIdx.x * blockDim.x + threadIdx.x;
    // Initialize RNG state for this thread with a unique sequence number
    hiprand_init(seed, tidx, 0, &state[tidx]);
}