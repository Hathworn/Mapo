#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Update kernel to perform atomic division for thread-safe operation
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        atomicExch(accuracy, *accuracy / N);
    }
}