#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void double_value(double *x, double *y)
{
    // Compute global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Double the value if within bounds
    if (index < 1024) // Assumes a predefined size, replace 1024 with actual size if needed
    {
        y[index] = 2.0 * x[index];
    }
}