#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Not(bool* x, size_t idx, size_t N)
{
    // Calculate the index this thread will operate on
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Increase stride to cover all elements
    int stride = blockDim.x * gridDim.x;
    
    // Loop with stride to operate on all necessary elements
    for (; i < N; i += stride)
    {
        // Toggle the boolean value at the current index
        x[(idx - 1) * N + i] = !x[(idx - 1) * N + i];
    }
}