#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void IfThenElse(bool * b, float * x, size_t idxb, size_t idxf, size_t N)
{
    // Calculate global index once instead of inside loop
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid unnecessary calculations inside the loop
    size_t b_offset = (idxb-1) * N;
    size_t x_offset_1 = (idxf-1) * N;
    size_t x_offset_2 = (idxf-2) * N;

    // Loop to handle all elements assigned to this thread
    for (; i < N; i += blockDim.x * gridDim.x)
    {
        // Simplified condition and assignment
        if (b[b_offset + i])
            x[x_offset_2 + i] = x[x_offset_1 + i];
    }
}