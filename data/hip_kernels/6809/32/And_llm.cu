#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void And(bool *x, size_t idx, size_t N)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a stride loop to handle larger N efficiently
    int stride = blockDim.x * gridDim.x;
    for (; i < N; i += stride)
    {
        x[(idx - 2) * N + i] = x[(idx - 2) * N + i] & x[(idx - 1) * N + i];
    }
}