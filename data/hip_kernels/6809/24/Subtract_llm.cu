#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract( float * x, size_t idx, size_t N, float W0, float W1)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop unrolling through stride
    const int stride = blockDim.x * gridDim.x;

    // Optimize by processing multiple elements per thread with coalesced memory access
    for (; i < N; i += stride)
    {
        float val1 = x[(idx-1)*N + i];
        float val2 = x[(idx-2)*N + i];
        x[(idx-2)*N + i] = W0 * val1 - W1 * val2;
    }
}