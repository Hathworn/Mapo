#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GEQ(float * xf, bool * xb, size_t idxf, size_t idxb, size_t N)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride to ensure each thread processes multiple elements if necessary
    int stride = blockDim.x * gridDim.x;

    // Loop over elements with stride for efficient memory access
    for (; i < N; i += stride)
    {
        xb[idxb*N+i] = xf[(idxf-1)*N+i] >= xf[(idxf-2)*N+i];
    }
}