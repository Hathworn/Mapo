#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Add(float * x, size_t idx, size_t N, float W0, float W1)
{
    // Optimize thread stride loop to avoid repeated arithmetic calculations
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int step = blockDim.x * gridDim.x;
    int offset1 = (idx-1) * N;
    int offset2 = (idx-2) * N;

    for (; i < N; i += step)
    {
        // Reduce the array index calculation inside the loop
        x[offset2 + i] = x[offset1 + i] * W0 + x[offset2 + i] * W1;
    }
}