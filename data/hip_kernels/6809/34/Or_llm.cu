#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Or(bool * x, size_t idx, size_t N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (; i < N; i += stride)
    {
        // Optimize memory access with logical operation
        x[(idx-2)*N+i] |= x[(idx-1)*N+i];
    }
}