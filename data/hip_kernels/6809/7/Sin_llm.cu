#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Sin(float *x, size_t idx, size_t N, float W0)
{
    // Cache computed index calculations for efficiency
    size_t baseIdx = (idx - 1) * N;
    int threadStride = blockDim.x * gridDim.x;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += threadStride)
    {
        x[baseIdx + i] = sin(W0 * x[baseIdx + i]);
    }
}