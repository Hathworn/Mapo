#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Cos(float *x, size_t idx, size_t N, float W0)
{
    // Calculate the global index of the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread works on elements within bounds
    if (i < N)
    {
        x[(idx-1)*N+i] = cos(W0 * x[(idx-1)*N+i]);
    }
}