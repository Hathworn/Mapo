#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ConstantB(bool * x, bool value, size_t idx, size_t N)
{
    // Compute global index once outside of loop
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory if beneficial (example use of shared memory not shown due to brevity)
    // __shared__ bool sharedArray[BLOCK_SIZE]; 

    for (int i = globalIndex; i < N; i += stride)
    {
        x[idx * N + i] = value;
    }
}