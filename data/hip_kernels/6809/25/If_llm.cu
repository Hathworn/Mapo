#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void If(bool * xb, float * xf, size_t idxf, size_t idxb, size_t N)
{
    // Calculate global index once to avoid recalculating in the loop
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = globalIdx; i < N; i += stride)
    {
        // Use shared memory or registers if more complex computation is needed
        if (!xb[(idxb-1)*N+i])
            xf[(idxf-1)*N+i] = 0;
    }
}