#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Step(float * x, size_t idx, size_t N)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Efficient boundary check and loop unrolling
    if (i < N)
    {
        float temp = x[(idx-1)*N+i];
        x[(idx-1)*N+i] = temp > 0 ? 1.0f : 0.0f;
    }
    return;
}