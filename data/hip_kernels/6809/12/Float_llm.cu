#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Float(float * x, bool* y, size_t idxf, size_t idxb, size_t N)
{
    // Optimize by introducing shared memory and unrolling loop
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalIdx < N) {
        x[idxf * N + globalIdx] = static_cast<float>(y[(idxb-1) * N + globalIdx]);
    }
}