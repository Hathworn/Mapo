#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Xor( bool * x, size_t idx, size_t N)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop to optimize memory access
    int stride = blockDim.x * gridDim.x;
    while (i < N)
    {
        x[(idx-2)*N+i] = x[(idx-1)*N+i] != x[(idx-2)*N+i];
        i += stride;
    }
}