#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Split(float * xf, bool * xb, size_t idxf, size_t idxb, size_t N, float threshold)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure thread does not exceed N
    if (i < N)
    {
        // Perform threshold check and assignment
        xb[(idxb) * N + i] = (xf[(idxf - 1) * N + i] < threshold);
    }
}
```
