#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Divide(float *x, size_t idx, size_t N, float W0, float W1)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;

    // Loop with striding for better GPU utilization
    for (; i < N; i += step)
    {
        float denominator = x[(idx-2)*N+i];
        if (abs(denominator) > 1e-8) // Use scientific notation for better readability
        {
            x[(idx-2)*N+i] = (W0 * x[(idx-1)*N+i]) / (W1 * denominator);
        }
        else
        {
            x[(idx-2)*N+i] = 1.0f; // Use '1.0f' for float consistency
        }
    }
}