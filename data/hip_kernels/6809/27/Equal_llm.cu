#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Equal(float * xf, bool * xb, size_t idxf, size_t idxb, size_t N)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll loop to improve performance
    if (i < N)
    {
        xb[(idxb)*N + i] = xf[(idxf-1)*N + i] == xf[(idxf-2)*N + i];
    }
}