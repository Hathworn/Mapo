#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LEQ(float * xf, bool * xb, size_t idxf, size_t idxb, size_t N)
{
    // Cache the grid size for reduced computation
    int gridSize = blockDim.x * gridDim.x;
    // Utilize shared memory for repeated data access within a block
    extern __shared__ float shared_xf[];
    
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += gridSize)
    {
        // Load elements that will be accessed more than once into shared memory
        shared_xf[threadIdx.x] = xf[(idxf - 2) * N + i];
        __syncthreads();

        // Perform the comparison using shared memory
        if (i < N) // Additional check to ensure within bounds
        {
            xb[idxb * N + i] = xf[(idxf - 1) * N + i] <= shared_xf[threadIdx.x];
        }
        __syncthreads();
    }
}