#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GreaterThan(float * xf, bool * xb, size_t idxf, size_t idxb, size_t N)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds of the data
    if (i < N)
    {
        // Perform comparison operation
        xb[idxb*N + i] = xf[(idxf-1)*N + i] > xf[(idxf-2)*N + i];
    }
}