#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void Sqrt(float *x, size_t idx, size_t N, float W0) {
    // Use shared memory for coefficients if needed in larger context        
    size_t stride = blockDim.x * gridDim.x;
    size_t offset = (idx - 1) * N;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += stride) {
        float val = x[offset + i];
        x[offset + i] = sqrtf(fabsf(val) * W0);  // Use sqrtf and fabsf for better performance
    }
}