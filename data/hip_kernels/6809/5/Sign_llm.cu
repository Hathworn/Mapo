#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Sign(float *x, size_t idx, size_t N)
{
    // Calculate the global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a while loop instead of for-loop for better readability
    while (i < N)
    {
        // Use ternary operator for compact conditional assignment
        float res = x[(idx - 1) * N + i];
        x[(idx - 1) * N + i] = (res > 0) ? 1.0 : (res == 0) ? 0.0 : -1.0;

        // Increment by the total number of threads
        i += blockDim.x * gridDim.x;
    }
}
```
