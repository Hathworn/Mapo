#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Split(int * xi, bool * xb, size_t idxi, size_t idxb, size_t N, float threshold)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a loop with a step equal to the grid size for coalesced memory access
    for (; i < N; i += blockDim.x * gridDim.x)
    {
        // Direct access to the specific index
        xb[(idxb)*N + i] = (((float)xi[(idxi - 1) * N + i]) == threshold);
    }
}