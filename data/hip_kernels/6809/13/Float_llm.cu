#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Float(float* x, int* y, size_t idxf, size_t idxi, size_t N)
{
    // Calculate the global index of the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a loop if i is less than N
    while (i < N) {
        // Directly access and assign values with optimized indexing
        x[idxf * N + i] = static_cast<float>(y[(idxi - 1) * N + i]);
        
        // Increment by the total number of threads
        i += blockDim.x * gridDim.x;
    }
}