#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Relu(float *x, size_t idx, size_t N, float W0)
{
    // Calculate linear thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) { // Check to avoid processing beyond array bounds
        // Perform ReLU operation with a check to reduce computation    
        float val = W0 * x[(idx - 1) * N + i];
        x[(idx - 1) * N + i] = val > 0 ? val : 0.01;
    }
}