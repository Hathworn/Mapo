#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LessThan(float * xf, bool * xb, size_t idxf, size_t idxb, size_t N)
{
    // Using shared memory for better memory access patterns
    extern __shared__ float sharedMem[];

    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load elements into shared memory within bounds
    if (i < N) {
        sharedMem[threadIdx.x] = xf[(idxf-1)*N + i];
        sharedMem[threadIdx.x + blockDim.x] = xf[(idxf-2)*N + i];
    }
    __syncthreads();

    // Perform comparison using shared memory within bounds
    if (i < N) {
        xb[idxb*N + i] = (sharedMem[threadIdx.x] < sharedMem[threadIdx.x + blockDim.x]);
    }
}