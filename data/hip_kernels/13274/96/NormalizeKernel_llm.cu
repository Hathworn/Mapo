#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Early exit for threads out of range
    if (i >= h || j >= w) return;

    const int pos = i * s + j;

    float scale = normalization_factor[pos];

    // Use ternary operator to handle zero scale
    image[pos] *= (scale != 0.0f) ? (1.0f / scale) : 1.0f;
}