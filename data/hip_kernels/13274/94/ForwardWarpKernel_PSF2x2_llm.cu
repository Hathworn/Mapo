#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Precompute variables to avoid redundant calculations
    float cx = u[flow_row_offset + j] * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + static_cast<float>(i) + 1.0f;

    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    
    // Target pixel integer coordinates
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);
    float value = src[image_row_offset + j];

    // Combine conditions to reduce code duplication and improve readability
    for (int x_offset = 0; x_offset <= 1; ++x_offset) {
        for (int y_offset = 0; y_offset <= 1; ++y_offset) {
            int tx_offset = tx - x_offset;
            int ty_offset = ty - y_offset;
            if (tx_offset >= 0 && tx_offset < w && ty_offset >= 0 && ty_offset < h) {
                float weight = ((x_offset == 0) ? (1.0f - dx) : dx) * ((y_offset == 0) ? (1.0f - dy) : dy);
                atomicAdd(dst + ty_offset * image_stride + tx_offset, value * weight);
                atomicAdd(normalization_factor + ty_offset * image_stride + tx_offset, weight);
            }
        }
    }
}
```
