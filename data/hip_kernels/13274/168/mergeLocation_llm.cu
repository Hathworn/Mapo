#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll condition to avoid warp divergence
    if (ptidx >= npoints) return;
    
    short2 loc = loc_[ptidx];

    // Utilize registers for calculations
    float scaled_x = loc.x * scale;
    float scaled_y = loc.y * scale;
    
    // Write results back to global memory
    x[ptidx] = scaled_x;
    y[ptidx] = scaled_y;
}