#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixDivisionScalar(double *a, double b, double *c, int cr, int cc) {

    // Calculate unique global thread index
    int idx = blockIdx.y * blockDim.y * cc + blockIdx.x * blockDim.x + threadIdx.y * cc + threadIdx.x;

    // Check index within bounds before processing
    if(idx < cr * cc) {
        c[idx] = a[idx] / b;
    }
}