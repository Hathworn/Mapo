#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixClip(double *a, double min, double max, double *c, int cr, int cc){

    int x = blockIdx.x * blockDim.x + threadIdx.x; // col
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    if(x < cc && y < cr){
        // Use min and max functions for concise conditional logic
        double value = a[y * cc + x];
        value = fmax(min, fmin(max, value));
        c[y * cc + x] = value;
    }

}