#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixSubScalarInverse(double *a, double b, double *c, int cr, int cc){

    // Calculate global thread ID in matrix
    int index = blockIdx.y * blockDim.y * cc + blockIdx.x * blockDim.x + threadIdx.y * cc + threadIdx.x;

    // Ensure work is only done within valid data range
    if(index < cr * cc){
        c[index] = b - a[index];
    }
}