#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixTranspose(double *a, double *c, int cr, int cc) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // col
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    // Check if the current thread is within bounds
    if (x < cc && y < cr) {
        // Optimize with coalesced access pattern
        int rowStart = y * cc;
        int colStart = x * cr;
        c[rowStart + x] = a[colStart + y];
    }
}