#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixSubScalar(double *a, double b, double *c, int cr, int cc){

    int idx = blockIdx.y * blockDim.y * cc + blockIdx.x * blockDim.x; // Calculate starting index
    int thread_id = threadIdx.y * blockDim.x + threadIdx.x; // Flatten thread index
    int x = idx + thread_id; // Overall 1D indexing

    if (x < cr * cc) { // Check within bounds for flattened index
        c[x] = a[x] - b; // Simplified the subtraction operation
    }

}