#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixSub(double *a, double *b, double *c, int cr, int cc) {
    // Calculate unique index for each thread
    long index = blockIdx.y * blockDim.y * cc + threadIdx.y * cc + blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread operates within matrix bounds
    if (index < cr * cc) {
        c[index] = a[index] - b[index];
    }
}