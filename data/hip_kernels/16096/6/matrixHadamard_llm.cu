#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixHadamard(double *a, double *b, double *c, int cr, int cc) {
    // Use shared memory for improved memory access
    extern __shared__ double shared_mem[];
    double *shared_a = shared_mem;
    double *shared_b = &shared_mem[cc];

    long x = blockIdx.x * blockDim.x + threadIdx.x;
    long y = blockIdx.y * blockDim.y + threadIdx.y;

    // Load data into shared memory
    if (x < cc && y < cr) {
        shared_a[threadIdx.y * blockDim.x + threadIdx.x] = a[y * cc + x];
        shared_b[threadIdx.y * blockDim.x + threadIdx.x] = b[y * cc + x];
    }
    __syncthreads();

    // Perform computation using shared memory
    if (x < cc && y < cr) {
        c[y * cc + x] = shared_a[threadIdx.y * blockDim.x + threadIdx.x] * shared_b[threadIdx.y * blockDim.x + threadIdx.x];
    }
}