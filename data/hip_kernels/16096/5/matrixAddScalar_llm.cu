#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixAddScalar(double *a, double b, double *c, int cr, int cc){
    // Calculate global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x; // col
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    // Index calculation outside the condition to avoid recalculation
    int idx = y * cc + x;

    // Ensure threads within matrix boundaries perform computations
    if(x < cc && y < cr) {
        c[idx] = a[idx] + b; // Efficient element access
    }
}