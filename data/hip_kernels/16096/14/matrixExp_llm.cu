#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixExp(double *a, double *c, int cr, int cc) {
    // Calculate global thread ID for both row and column
    int idx = blockIdx.y * blockDim.y + threadIdx.y; // row
    int idy = blockIdx.x * blockDim.x + threadIdx.x; // col

    // Ensure index is within bounds and perform the operation
    if (idx < cr && idy < cc) {
        int index = idx * cc + idy;
        c[index] = exp(a[index]);
    }
}