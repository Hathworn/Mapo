#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixAdd(double *a, double *b, double *c, int cr, int cc){
    // Calculate the flat index for the 1D grid
    long idx = blockIdx.y * blockDim.y * cc + blockIdx.x * blockDim.x + threadIdx.y * cc + threadIdx.x;
    
    // Perform sum if within bounds
    if(idx < cr * cc){
        c[idx] = a[idx] + b[idx];
    }  
}