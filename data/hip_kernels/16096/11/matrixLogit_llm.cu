#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixLogit(double *a, double *c, int cr, int cc) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // col
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    // Use private variable to reduce memory access and improve performance
    if (x < cc && y < cr) {
        double a_val = a[y * cc + x];
        c[y * cc + x] = log(a_val / (1 - a_val));
    }
}