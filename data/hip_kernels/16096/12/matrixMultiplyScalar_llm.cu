#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMultiplyScalar(double *a, double b, double *c, int cr, int cc) {
    // Calculate a single index instead of x and y separately
    int index = blockIdx.y * blockDim.y * cc + threadIdx.y * cc + blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single barrier for range check
    if (index < cr * cc) {
        c[index] = a[index] * b;
    }
}