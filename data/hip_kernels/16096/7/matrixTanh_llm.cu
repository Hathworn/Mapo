#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixTanh(double *a, double *c, int cr, int cc){

    int x = blockIdx.x * blockDim.x + threadIdx.x; // col
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    if(x < cc && y < cr){
        // Use shared memory to reduce global memory access
        extern __shared__ double shared_a[];
        int index = y * cc + x;
        shared_a[threadIdx.y * blockDim.x + threadIdx.x] = a[index];
        __syncthreads();

        // Compute tanh using shared memory
        c[index] = tanh(shared_a[threadIdx.y * blockDim.x + threadIdx.x]);
    }
}