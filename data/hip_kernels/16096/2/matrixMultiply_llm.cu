#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMultiply(double *a, double *b, double *c, int cr, int cc, int ac, int bc) {
    // Use shared memory for better memory access coalescing
    __shared__ double shared_a[TILE_SIZE][TILE_SIZE];
    __shared__ double shared_b[TILE_SIZE][TILE_SIZE];

    // Calculate row and column index of the element
    long x = blockIdx.x * blockDim.x + threadIdx.x; // col
    long y = blockIdx.y * blockDim.y + threadIdx.y; // row
    double sum = 0;

    // Iterate over tiles
    for (int t = 0; t < (ac + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Load data into shared memory
        if (y < cr && t * TILE_SIZE + threadIdx.x < ac) {
            shared_a[threadIdx.y][threadIdx.x] = a[y * ac + t * TILE_SIZE + threadIdx.x];
        } else {
            shared_a[threadIdx.y][threadIdx.x] = 0.0;
        }
        
        if (x < cc && t * TILE_SIZE + threadIdx.y < ac) {
            shared_b[threadIdx.y][threadIdx.x] = b[(t * TILE_SIZE + threadIdx.y) * bc + x];
        } else {
            shared_b[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads(); // Synchronize threads to ensure complete tile load

        // Accumulate partial result
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += shared_a[threadIdx.y][k] * shared_b[k][threadIdx.x];
        }

        __syncthreads(); // Synchronize threads before loading new tile
    }

    // Write result to the matrix
    if (x < cc && y < cr) {
        c[y * cc + x] = sum;
    }
}