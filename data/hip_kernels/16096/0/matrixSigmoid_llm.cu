#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixSigmoid(double *a, double *c, int cr, int cc) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // col index
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row index

    // Calculate the linear index and perform the operation if within bounds
    int idx = y * cc + x;
    if (x < cc && y < cr) {
        // Direct computation using pre-calculated index
        c[idx] = 1.0 / (1 + exp(-a[idx]));
    }
}