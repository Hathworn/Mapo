#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Use a loop to process multiple elements per thread if possible
    int stride = gridDim.x * blockDim.x;
    for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < N; tid += stride) {
        out[tid] = fmaxf(in[tid], 0.0f); // Use fmaxf for potential performance benefit
    }
}