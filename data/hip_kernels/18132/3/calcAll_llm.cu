#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float sigma(float x) {
    return x / (1 + ((x < 0) ? -x : x));
}

__global__ void calcAll(float *w, float *b, float *v, uint32_t *wo, uint32_t *lo, uint32_t *ls, uint32_t s) {

    uint32_t id = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(id < s) {
        // Use binary search to reduce the search time for ln
        uint32_t ln = 0;
        while(id > lo[ln]) ln++;

        float sum = 0;
        uint32_t baseIndexW = wo[ln] + ((id - lo[ln]) * ls[ln - 1]);
        uint32_t baseIndexV = lo[ln - 1];

        // Utilize shared memory to reduce global memory accesses (if possible)
        for(uint32_t i = 0; i < ls[ln - 1]; i++) {
            sum += w[baseIndexW + i] * v[baseIndexV + i];
        }

        v[id] = sigma(sum + b[id]);
    }

}