#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initMult(uint32_t *A, uint32_t size, uint32_t mult) {
    uint32_t id = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t stride = blockDim.x * gridDim.x; // Calculate the stride for loop unrolling
    for (uint32_t i = id; i < size; i += stride) {
        A[i] = i * mult; // Unroll kernel to handle multiple elements per thread
    }
}