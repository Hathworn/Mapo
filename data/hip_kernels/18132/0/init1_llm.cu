#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init1(uint32_t *A, uint32_t size) {
    uint32_t id = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    // Use loop unrolling for performance improvement
    if (id < size) {
        A[id] = 1;
    }
    if (id + blockDim.x < size) {
        A[id + blockDim.x] = 1;
    }
    if (id + 2 * blockDim.x < size) {
        A[id + 2 * blockDim.x] = 1;
    }
    if (id + 3 * blockDim.x < size) {
        A[id + 3 * blockDim.x] = 1;
    }
}