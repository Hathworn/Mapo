#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initn(uint32_t *A, uint32_t size, uint32_t n) {
    uint32_t id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use loop unrolling for better parallelism
    for (uint32_t offset = 0; id + offset < size; offset += blockDim.x * gridDim.x) {
        A[id + offset] = n;
    }
}