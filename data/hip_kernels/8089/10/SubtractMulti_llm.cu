#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Compute x and y coordinates
    const int x = blockIdx.x * SUBTRACTM_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACTM_H + threadIdx.y;
    
    // Compute index within the 3D data layout
    const int pitch_height = pitch * height;
    const int p = threadIdx.z * pitch_height + y * pitch + x;
    
    // Check within bounds and perform subtraction
    if (x < width && y < height) 
    {
        d_Result[p] = d_Data[p] - d_Data[p + pitch_height];
    }
}