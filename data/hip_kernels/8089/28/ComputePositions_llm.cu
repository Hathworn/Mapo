#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define POSBLK_SIZE 256

__global__ void ComputePositions(float *g_Data1, float *g_Data2, float *g_Data3, int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use simpler index calculation
    if (i >= numPts) return;

    int p = d_Ptrs[i];

    float val0 = g_Data2[p]; // Cache repeated memory accesses
    float val1 = g_Data2[p - 1];
    float val2 = g_Data2[p + 1];
    float dx = 0.5f * (val2 - val1);
    float dxx = 2.0f * val0 - val1 - val2;

    float val3 = g_Data2[p - w];
    float val4 = g_Data2[p + w];
    float dy = 0.5f * (val4 - val3);
    float dyy = 2.0f * val0 - val3 - val4;

    float val5 = g_Data3[p];
    float val6 = g_Data1[p];
    float ds = 0.5f * (val6 - val5);
    float dss = 2.0f * val0 - val5 - val6;

    float dxy = 0.25f * (g_Data2[p + w + 1] + g_Data2[p - w - 1] - g_Data2[p - w + 1] - g_Data2[p + w - 1]);
    float dxs = 0.25f * (g_Data3[p + 1] + g_Data1[p - 1] - g_Data1[p + 1] - g_Data3[p - 1]);
    float dys = 0.25f * (g_Data3[p + w] + g_Data1[p - w] - g_Data3[p - w] - g_Data1[p + w]);

    float idxx = dyy * dss - dys * dys;
    float idxy = dys * dxs - dxy * dss;
    float idxs = dxy * dys - dyy * dxs;
    float idyy = dxx * dss - dxs * dxs;
    float idys = dxy * dxs - dxx * dys;
    float idss = dxx * dyy - dxy * dxy;

    float det = idxx * dxx + idxy * dxy + idxs * dxs;
    float idet = 1.0f / det;

    float pdx = idet * (idxx * dx + idxy * dy + idxs * ds);
    float pdy = idet * (idxy * dx + idyy * dy + idys * ds);
    float pds = idet * (idxs * dx + idys * dy + idss * ds);

    if (pdx < -0.5f || pdx > 0.5f || pdy < -0.5f || pdy > 0.5f || pds < -0.5f || pds > 0.5f) {
        pdx = dx / dxx;
        pdy = dy / dyy;
        pds = ds / dss;
    }

    float dval = 0.5f * (dx * pdx + dy * pdy + ds * pds);

    d_Sift[i + 0 * maxPts] = (p % w) + pdx;
    d_Sift[i + 1 * maxPts] = (p / w) + pdy;
    d_Sift[i + 2 * maxPts] = d_ConstantA[0] * exp2f(pds * d_ConstantB[0]);
    d_Sift[i + 3 * maxPts] = val0 + dval;

    float tra = dxx + dyy;
    det = dxx * dyy - dxy * dxy;
    d_Sift[i + 4 * maxPts] = (tra * tra) / det;
}