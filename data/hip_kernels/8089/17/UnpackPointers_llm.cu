#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    const int blockSize = blockDim.x;

    int numPts = 0;
    for (int y = 0; y < h / 32; y++) {
        for (int x = tx; x < w; x += blockSize) { // optimize work distribution by blocks
            unsigned int val = minmax[y * w + x];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        int index = atomicAdd(&numPts, 1); // atomic operation for shared counter
                        if (index < maxPts) { // double check within bounds
                            ptrs[16 * index + tx] = (y * 32 + k) * w + x;
                        }
                    }
                    val >>= 1;
                }
            }
        }
    }
}