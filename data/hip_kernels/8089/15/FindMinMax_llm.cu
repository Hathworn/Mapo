#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    const int b = blockDim.x;
    int p = y * pitch + x;

    // Initialize shared memory with boundary checks
    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
        for (int ty = 1; ty < 16; ty++) {
            p += pitch;
            val = d_Data[p];
            if (val < minvals[tx]) minvals[tx] = val;
            if (val > maxvals[tx]) maxvals[tx] = val;
        }
    } else {
        minvals[tx] = FLT_MAX;
        maxvals[tx] = FLT_MIN;
    }
    
    // Synchronize threads before reduction
    __syncthreads();

    // Reduction in shared memory
    for (int stride = b / 2; stride > 0; stride >>= 1) {
        if (tx < stride) {
            if (minvals[tx + stride] < minvals[tx]) {
                minvals[tx] = minvals[tx + stride];
            }
            if (maxvals[tx + stride] > maxvals[tx]) {
                maxvals[tx] = maxvals[tx + stride];
            }
        }
        __syncthreads();
    }

    // Write the result from block to global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr + 0] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}