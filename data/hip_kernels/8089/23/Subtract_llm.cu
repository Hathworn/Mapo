#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Use blockDim for thread indexing
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int p = y * width + x;

    // Check if within bounds
    if (x < width && y < height)
    {
        // Perform subtraction
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}