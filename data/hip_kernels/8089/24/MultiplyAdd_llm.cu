#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Optimized calculation of x coordinate
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Optimized calculation of y coordinate
    int p = y * width + x; // Optimized calculation of linear index
    if (x < width && y < height) // Check if within bounds
    {
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0]; // Perform multiply-add operation
    }
    __syncthreads(); // Synchronize threads within the block
}