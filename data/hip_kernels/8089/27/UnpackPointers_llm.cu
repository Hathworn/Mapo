#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    // Use shared memory for efficient access
    extern __shared__ int sharedMinmax[];

    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    const int stride = blockDim.x;

    int numPts = 0;

    for (int y = bx; y < h / 32; y += gridDim.x) {
        for (int x = 0; x < w; x += stride) {
            // Load data into shared memory for coalesced access
            sharedMinmax[tx] = (x + tx < w) ? minmax[y * w + x + tx] : 0;
            __syncthreads();

            unsigned int val = sharedMinmax[tx];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if ((val & 0x1) && (numPts < maxPts)) {
                        // Write result to global memory with coalesced access  
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
            __syncthreads();
        }
    }
}