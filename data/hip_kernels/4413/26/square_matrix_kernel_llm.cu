#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Calculate row and column indices for the current thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Only proceed if the indices are within matrix bounds
    if (row < num_rows && col < num_cols) {
        float f = feats[row * ldf + col]; // Access the feature value
        feats_sq[row * lds + col] = f * f; // Store squared value
    }
}