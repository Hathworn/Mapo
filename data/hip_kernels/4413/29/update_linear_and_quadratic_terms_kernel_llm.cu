#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel(int32_t n, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    // Calculate new scale and update val if max_count is valid
    float val = 1.0f;
    float cur_weight = *cur_tot_weight;
    if (max_count > 0.0f) {
        float new_scale = max(cur_weight, (float)max_count) / max_count;
        val += new_scale - 1.0f;
    }
    
    // Calculate index and update quadratic values for each thread
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride = blockDim.x * gridDim.x;
    for (int32_t i = idx; i < n; i += stride) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        atomicAdd(&quadratic[diag_idx], val); // Use atomicAdd to ensure correctness in concurrent updates
    }

    // Update linear term by one thread in each block
    if (threadIdx.x == 0) {
        atomicAdd(&linear[0], val * prior_offset); // Use atomicAdd for shared memory write
    }
}