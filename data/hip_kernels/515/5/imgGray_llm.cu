#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void imgGray(unsigned char * d_image, unsigned char* d_imagegray, int width, int height){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width) {
        int grayIdx = row * width + col;
        int colorIdx = grayIdx * 3;
        
        // Perform weighted sum directly using local variables
        unsigned char r = d_image[colorIdx];
        unsigned char g = d_image[colorIdx + 1];
        unsigned char b = d_image[colorIdx + 2];
        
        d_imagegray[grayIdx] = r * 0.114f + g * 0.587f + b * 0.299f; // Optimized for readability and efficiency
    }
}