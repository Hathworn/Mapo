#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void imgGray(unsigned char * d_image, unsigned char* d_imagegray, int width, int height) {

    // Calculate the row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure pixel indices are within the image boundaries
    if ((width > col) && (height > row)) {
        // Pre-calculate index for image and grayscale
        int imgIndex = (row * width + col) * 3;
        int grayIndex = row * width + col;

        // Compute grayscale value using weighted sum and store
        d_imagegray[grayIndex] = d_image[imgIndex + 2] * 0.299f +
                                 d_image[imgIndex + 1] * 0.587f +
                                 d_image[imgIndex] * 0.114f;
    }
}