#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MultiplicarMatrices(float *m1, float *m2, float *mr, int columna1, int fila1, int columna2, int fila2)
{
    int fila_r = blockIdx.y * blockDim.y + threadIdx.y;
    int columna_r = blockIdx.x * blockDim.x + threadIdx.x;
    float tmp_mult = 0;

    // Optimize by checking condition before entering the loop
    if ((fila_r < fila1) && (columna_r < columna2))
    {
        // Unroll the loop partially for potential performance gains
        for (int i = 0; i < fila2; i += 2)
        {
            tmp_mult += m1[i + columna1 * fila_r] * m2[i * columna2 + columna_r];
            if (i + 1 < fila2)
            {
                tmp_mult += m1[(i + 1) + columna1 * fila_r] * m2[(i + 1) * columna2 + columna_r];
            }
        }
        mr[fila_r * columna2 + columna_r] = tmp_mult;
    }
}