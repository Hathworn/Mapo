#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_index = blockIdx.x;
    idx += m * nsample * batch_index;
    grad_out += m * nsample * c * batch_index;
    grad_points += n * c * batch_index;

    int index = blockDim.x * blockIdx.y + threadIdx.x; // Improved indexing with additional grid dimension
    int stride = blockDim.x * gridDim.y;

    for (int j = index; j < m * nsample; j += stride) { // Unrolled j-loop for better coalescing
        int sample_idx = j / nsample;
        int local_k = j % nsample;
        int ii = idx[sample_idx * nsample + local_k];
        for (int l = 0; l < c; ++l) {
            atomicAdd(&grad_points[ii * c + l], grad_out[sample_idx * nsample * c + local_k * c + l]);
        }
    }
}