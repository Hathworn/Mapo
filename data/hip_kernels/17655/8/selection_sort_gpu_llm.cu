#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    outi += m * n * batch_index;
    out += m * n * batch_index;
    
    int index = threadIdx.x;
    int stride = blockDim.x;
    
    // Optimize data copy to use shared memory
    __shared__ float shared_dist[1024];  // Assume maximum stride size
    __shared__ int shared_outi[1024];

    for (int j = index; j < m; j += stride) {
        for (int s = 0; s < n; ++s) {
            int idx = j * n + s;
            shared_dist[idx] = dist[idx];
            shared_outi[idx] = s;
        }
    }
    __syncthreads();

    float *p_dist;
    for (int j = index; j < m; j += stride) {
        p_dist = shared_dist + j * n;
        // Optimize selection sort with fewer swap operations
        for (int s = 0; s < k; ++s) {
            int min = s;
            for (int t = s + 1; t < n; ++t) {
                if (p_dist[t] < p_dist[min]) {
                    min = t;
                }
            }
            // Swap only if min index has changed
            if (min != s) {
                float tmp = p_dist[min];
                p_dist[min] = p_dist[s];
                p_dist[s] = tmp;
                int tmpi = shared_outi[j * n + min];
                shared_outi[j * n + min] = shared_outi[j * n + s];
                shared_outi[j * n + s] = tmpi;
            }
        }
    }

    // Copy shared memory back to global memory
    __syncthreads();
    for (int j = index; j < m; j += stride) {
        for (int s = 0; s < n; ++s) {
            int idx = j * n + s;
            out[idx] = shared_dist[idx];
            outi[idx] = shared_outi[idx];
        }
    }
}