#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n * c * batch_index;
    idx += m * nsample * batch_index;
    out += m * nsample * c * batch_index;
    
    int index = threadIdx.x + blockDim.x * blockIdx.y; // Use 2D grid for better parallelization
    int stride = blockDim.x * gridDim.y;
    
    for (int j = index; j < m * nsample; j += stride) { // Unroll nested loops for efficiency
        int j_idx = j / nsample;
        int k_idx = j % nsample;
        int ii = idx[j_idx * nsample + k_idx];
        for (int l = 0; l < c; ++l) {
            out[j_idx * nsample * c + k_idx * c + l] = points[ii * c + l];
        }
    }
}