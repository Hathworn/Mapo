#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtractCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if the thread is within the bounds
    if (x < width && y < height) {
        const uint8_t *in1X = in1 + y * rowSizeIn1 + x;
        const uint8_t *in2X = in2 + y * rowSizeIn2 + x;
        uint8_t *outX = out + y * rowSizeOut + x;
        
        //Perform subtraction only if in1X > in2X, else set to 0
        *outX = ((*in1X) > (*in2X)) ? ((*in1X) - (*in2X)) : 0;
    }
}