#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void absoluteDifferenceCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        // Load values from global memory once per thread
        const uint8_t val1 = in1[y * rowSizeIn1 + x];
        const uint8_t val2 = in2[y * rowSizeIn2 + x];
        uint8_t result = (val1 > val2) ? (val1 - val2) : (val2 - val1);
        out[y * rowSizeOut + x] = result;
    }
}