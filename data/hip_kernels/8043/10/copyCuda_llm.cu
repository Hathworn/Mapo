#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Compute x and y coordinates
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if coordinates are within bounds before performing the copy
    if (x < width && y < height) {
        // Perform data copy
        out[y * rowSizeOut + x] = in[y * rowSizeIn + x];
    }
}