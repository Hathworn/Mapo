#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void minimumCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if ( x < width && y < height ) {
        const uint8_t in1Val = in1[y * rowSizeIn1 + x];
        const uint8_t in2Val = in2[y * rowSizeIn2 + x];
        out[y * rowSizeOut + x] = (in1Val < in2Val) ? in1Val : in2Val; // Use local variables to optimize memory access
    }
}