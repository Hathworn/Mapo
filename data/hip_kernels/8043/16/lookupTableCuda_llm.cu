#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lookupTableCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height, uint8_t *table)
{
    // Calculate x and y indices
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure within bounds
    if (x < width && y < height)
    {
        // Optimize memory access
        uint32_t inIdx = y * rowSizeIn + x;
        uint32_t outIdx = y * rowSizeOut + x;

        out[outIdx] = table[in[inIdx]];
    }
}