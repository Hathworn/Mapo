#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized bitwise XOR CUDA kernel
__global__ void bitwiseXorCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate thread positions
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Only execute within image boundaries
    if (x < width && y < height) {
        // Compute linear indices for input and output arrays
        const uint32_t idIn1 = y * rowSizeIn1 + x;
        const uint32_t idIn2 = y * rowSizeIn2 + x;
        const uint32_t idOut = y * rowSizeOut + x;
        // Perform bitwise XOR operation
        out[idOut] = in1[idIn1] ^ in2[idIn2];
    }
}