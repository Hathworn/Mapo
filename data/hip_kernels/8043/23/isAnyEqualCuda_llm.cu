#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void isAnyEqualCuda( const uint8_t * image, uint8_t * value, size_t valueCount, uint32_t width, uint32_t height, uint32_t * differenceCount )
{
    // Calculate the unique thread index
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Ensure the thread is within image bounds
    if (x >= width || y >= height) return;
    
    const uint32_t id = y * width + x;
    bool equal = false;
    
    // Iterate through values to check for matches
    for (uint32_t i = 0; i < valueCount; i++)
    {
        if (image[id] == value[i])
        {
            equal = true;
            break;  // Exit loop early if match is found
        }
    }
    
    // If any match was found, use atomic increment to update differenceCount
    if (equal)
    {
        atomicAdd(differenceCount, 1);
    }
}