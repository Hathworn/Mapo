#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogramCuda(const uint8_t *data, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t *histogram)
{
    // Calculate the unique thread index
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if the thread is within image bounds
    if (x < width && y < height) {
        // Calculate linear index in the data array
        const uint32_t id = y * rowSize + x;

        // Use atomicAdd for safe concurrent updates to the histogram array
        atomicAdd(&histogram[data[id]], 1);
    }
}