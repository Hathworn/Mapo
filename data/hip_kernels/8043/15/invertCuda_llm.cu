#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void invertCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate the pixel index
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure we are within the bounds of the image
    if (x < width && y < height)
    {
        // Perform the inversion
        out[y * rowSizeOut + x] = ~in[y * rowSizeIn + x];
    }
}