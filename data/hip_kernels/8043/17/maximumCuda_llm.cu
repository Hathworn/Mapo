#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void maximumCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if ( x < width && y < height ) {
        // Compute the input and output indices
        const uint32_t idx_in1 = y * rowSizeIn1 + x;
        const uint32_t idx_in2 = y * rowSizeIn2 + x;
        const uint32_t idx_out = y * rowSizeOut + x;

        // Perform the max operation directly on the elements
        out[idx_out] = max(in1[idx_in1], in2[idx_in2]);
    }
}