#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth(float * v_new, const float * v) {
    int myIdx = blockDim.x * blockIdx.x + threadIdx.x;  // Calculate global index
    int numThreads = gridDim.x * blockDim.x;

    // Use shared memory to reduce global memory access
    extern __shared__ float shared[];
    shared[threadIdx.x] = v[myIdx];

    __syncthreads();

    // Fetch neighbors using shared memory with boundary conditions
    float myElt = shared[threadIdx.x];
    float myLeftElt = (threadIdx.x == 0 && myIdx > 0) ? v[myIdx - 1] : ((threadIdx.x > 0) ? shared[threadIdx.x - 1] : myElt);
    float myRightElt = (threadIdx.x == blockDim.x - 1 && myIdx < numThreads - 1) ? v[myIdx + 1] : ((threadIdx.x < blockDim.x - 1) ? shared[threadIdx.x + 1] : myElt);

    // Calculating the new value
    v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
}