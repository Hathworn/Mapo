#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void naive_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    __shared__ int shared_bins[1024]; // Adjust size if necessary

    // Initialize shared memory
    if (threadIdx.x < BIN_COUNT) {
        shared_bins[threadIdx.x] = 0;
    }
    __syncthreads();

    // Calculate local histogram
    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;
    atomicAdd(&shared_bins[myBin], 1);
    __syncthreads();

    // Accumulate to global memory
    if (threadIdx.x < BIN_COUNT) {
        atomicAdd(&d_bins[threadIdx.x], shared_bins[threadIdx.x]);
    }
}