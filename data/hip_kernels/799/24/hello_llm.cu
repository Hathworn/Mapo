#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use a single thread to avoid multiple prints
    if (threadIdx.x == 0)
    {
        printf("Hello world! This is %d threads\n", blockDim.x);
    }
}