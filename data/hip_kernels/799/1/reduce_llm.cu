#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int *s) {
    int lane = threadIdx.x % 32;

    s[threadIdx.x] = p;
    __syncthreads();
    
    if (lane < 16) { s[threadIdx.x] += s[threadIdx.x + 16]; } __syncthreads();
    if (lane < 8)  { s[threadIdx.x] += s[threadIdx.x + 8]; }  __syncthreads();
    if (lane < 4)  { s[threadIdx.x] += s[threadIdx.x + 4]; }  __syncthreads();
    if (lane < 2)  { s[threadIdx.x] += s[threadIdx.x + 2]; }  __syncthreads();
    if (lane < 1)  { s[threadIdx.x] += s[threadIdx.x + 1]; }  __syncthreads();

    return s[threadIdx.x - lane]; // return result from lane 0 of each warp
}

__global__ void reduce(unsigned int *d_out_shared, const unsigned int *d_in)
{
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;

    // Perform warp-level reduction
    unsigned int sr = shared_reduce(d_in[t], s);

    // Aggregate results from all warps
    if (t % 32 == 0) {
        atomicAdd(d_out_shared, sr);
    }
}