#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void remove_redness_from_coordinates(const unsigned int* d_coordinates, unsigned char* d_r, unsigned char* d_b, unsigned char* d_g, unsigned char* d_r_output, int num_coordinates, int num_pixels_y, int num_pixels_x, int template_half_height, int template_half_width)
{
    int ny = num_pixels_y;
    int nx = num_pixels_x;
    int global_index_1d = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (global_index_1d < num_coordinates)
    {
        unsigned int image_index_1d = d_coordinates[global_index_1d]; // Changed to read forward through the coordinates.
        ushort2 image_index_2d = make_ushort2(image_index_1d % nx, image_index_1d / nx);

        int startY = max(0, image_index_2d.y - template_half_height); // Precomputed bounds for the loop.
        int endY = min(ny - 1, image_index_2d.y + template_half_height);
        int startX = max(0, image_index_2d.x - template_half_width);
        int endX = min(nx - 1, image_index_2d.x + template_half_width);

        for (int y = startY; y <= endY; y++)
        {
            for (int x = startX; x <= endX; x++)
            {
                int image_offset_index_1d_clamped = (nx * y) + x;

                unsigned char g_value = d_g[image_offset_index_1d_clamped];
                unsigned char b_value = d_b[image_offset_index_1d_clamped];

                unsigned int gb_average = (g_value + b_value) / 2;

                d_r_output[image_offset_index_1d_clamped] = (unsigned char)gb_average;
            }
        }
    }
}