#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// http://en.wikipedia.org/wiki/Bitonic_sort

__global__ void batcherBitonicMergesort64(float *d_out, const float *d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    for (int stage = 0; stage <= 5; ++stage)
    {
        int increment = 1 << stage;
        for (int substage = stage; substage >= 0; --substage)
        {
            // Efficient sorting within each stage and substage
            int subincrement = 1 << substage;
            int pairIdx = tid ^ subincrement;
            if (pairIdx > tid)
            {
                if ((tid & increment) == 0)
                {
                    if (sdata[tid] > sdata[pairIdx])
                    {
                        float temp = sdata[tid];
                        sdata[tid] = sdata[pairIdx];
                        sdata[pairIdx] = temp;
                    }
                }
                else
                {
                    if (sdata[tid] < sdata[pairIdx])
                    {
                        float temp = sdata[tid];
                        sdata[tid] = sdata[pairIdx];
                        sdata[pairIdx] = temp;
                    }
                }
            }
            __syncthreads(); // Synchronize within stages
        }
    }

    d_out[tid] = sdata[tid];
}