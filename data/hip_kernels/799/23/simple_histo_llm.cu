#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Get unique thread identifier
    int myId = threadIdx.x + blockDim.x * blockIdx.x;

    // Load input value
    int myItem = d_in[myId];
    
    // Compute bin index
    int myBin = myItem % BIN_COUNT;

    // Perform atomic addition
    atomicAdd(&(d_bins[myBin]), 1);
}