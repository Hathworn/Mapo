#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Each thread in the block prints its own unique statement
    printf("Hello world! I'm thread (%d, %d) in block (%d, %d)\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y);
}