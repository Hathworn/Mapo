#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int *d_input_cdf, float *d_output_cdf, int n)
{
    // Precompute the normalization constant outside the threads
    const float normalization_constant = 1.f / d_input_cdf[n - 1];

    // Compute 1D global index
    int global_index_1d = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Ensure the computed index is within bounds
    if (global_index_1d < n)
    {
        // Load input
        unsigned int input_value = d_input_cdf[global_index_1d];

        // Normalize and store
        d_output_cdf[global_index_1d] = input_value * normalization_constant;
    }
}