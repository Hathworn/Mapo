#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void global_reduce_kernel(float *d_out, float *d_in)
{
    extern __shared__ float sdata[]; // allocate shared memory
    
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    sdata[tid] = d_in[myId];  // load shared mem from global mem
    __syncthreads();          // ensure all loads complete

    // do reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();    // ensure all adds at one stage are done
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}