#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use CUDA warp efficiency where possible; unroll loops to minimize divergence
    if (tid < N) {
        float data = indata[tid];
        // Use ternary operator to avoid unnecessary branching
        ingrad[tid] = (data > 0) ? outgrad[tid] : 0;
    }
}