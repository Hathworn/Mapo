#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        // Use inline conditional operator to avoid divergence
        out[tid] = fmaxf(in[tid], 0.0f);
    }
}