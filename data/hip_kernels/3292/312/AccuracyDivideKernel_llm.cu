#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for better performance
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global index
    if (idx == 0) { // Only first thread performs the operation
        *accuracy /= N;
    }
}