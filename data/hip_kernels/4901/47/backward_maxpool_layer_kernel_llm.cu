#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int c = in_c;
    int area = (size - 1) / stride;

    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    int b = id / in_c; // Simplified calculation for b

    int w_offset = -pad;
    int h_offset = -pad;

    float d = 0;
    int out_w, out_h, out_index, valid;
    #pragma unroll // Unroll loop for potential performance gain
    for (int l = -area; l < area + 1; ++l) {
        #pragma unroll // Unroll loop for potential performance gain
        for (int m = -area; m < area + 1; ++m) {
            out_w = (j - w_offset) / stride + m;
            out_h = (i - h_offset) / stride + l;
            out_index = out_w + w * (out_h + h * (k + c * b));
            
            valid = (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h);
            d += (valid && indexes[out_index] == (b * in_c * in_h * in_w + k * in_h * in_w + i * in_w + j)) ? delta[out_index] : 0;
        }
    }
    prev_delta[b * in_c * in_h * in_w + k * in_h * in_w + i * in_w + j] += d; // Simplified index calculation
}