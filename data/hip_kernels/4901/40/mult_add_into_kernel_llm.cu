#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate global thread index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Process multiple elements per thread
    for (; i < n; i += stride) {
        c[i] += a[i] * b[i];
    }
}