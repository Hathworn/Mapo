#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Compute indices and offsets for input and output
    int in_index = i;
    int in_w = i % w;
    i /= w; // Simplified division
    int in_h = i % h;
    i /= h; // Simplified division
    int in_c = i % c;
    i /= c; // Simplified division
    int b = i; // Direct assignment since i is now the batch index

    int out_c = c / (stride * stride);

    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;

    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    // Assign value based on forward flag
    if (forward) {
        out[out_index] = x[in_index];
    } else {
        out[in_index] = x[out_index];
    }
}