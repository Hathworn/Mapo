#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation assuming 1D grid
    if(i < n){
        // Use shared memory for reused values
        __shared__ float shared_a, shared_b, shared_s, shared_dc;
        if (threadIdx.x == 0) {
            shared_a = a[blockIdx.x * blockDim.x];  // load block-specific value
            shared_b = b[blockIdx.x * blockDim.x];  // load block-specific value
            shared_s = s[blockIdx.x * blockDim.x];  // load block-specific value
            shared_dc = dc[blockIdx.x * blockDim.x];  // load block-specific value
        }
        __syncthreads();  // Synchronize threads within block

        // Perform operations using shared memory values
        if(da) da[i] += shared_dc * shared_s;
        db[i] += shared_dc * (1 - shared_s);
        ds[i] += shared_dc * shared_a + shared_dc * -shared_b;
    }
}