#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forward_softmax_layer_kernel( int n, int batch, float *input, float temp, float *output )
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if( b >= batch ) return;

    // Reduce memory access by using shared memory for input and output
    extern __shared__ float shared_mem[];
    float* shared_input = shared_mem;
    float* shared_output = shared_mem + n;

    // Load input to shared memory
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        shared_input[i] = input[i + b*n];
    }
    __syncthreads();

    float sum = 0;
    float largest = -INFINITY;

    // Find the largest element
    for( int i = 0; i < n; ++i ) {
        float val = shared_input[i];
        largest = fmaxf(val, largest);
    }

    // Compute sum and avoid recalculating exp(largest/temp)
    for( int i = 0; i < n; ++i ) {
        sum += expf(shared_input[i] / temp - largest / temp);
    }
    sum = (sum != 0) ? largest / temp + logf(sum) : largest - 100;

    // Calculate the softmax output
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        shared_output[i] = expf(shared_input[i] / temp - sum);
        output[i + b*n] = shared_output[i];
    }
}