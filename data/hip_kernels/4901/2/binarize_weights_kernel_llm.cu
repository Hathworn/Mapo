#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Utilize shared memory to calculate mean more efficiently
    extern __shared__ float shared_weights[];
    float mean = 0;
    
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[i] = abs(weights[f * size + i]);
    }
    __syncthreads();

    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        atomicAdd(&mean, shared_weights[i]);
    }
    __syncthreads();

    mean = mean / size;

    // Binarize weights with mean
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    }
}