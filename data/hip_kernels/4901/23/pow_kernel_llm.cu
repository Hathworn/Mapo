#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D grid
    int stride = gridDim.x * blockDim.x; // Calculate stride
    for (; i < N; i += stride) { // Loop over data with stride
        Y[i * INCY] = powf(X[i * INCX], ALPHA); // Use powf for float
    }
}