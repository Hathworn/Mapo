#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation
    int stride = gridDim.x * blockDim.x; // Calculate stride for loop unrolling
    for (int i = idx; i < N; i += stride) { // Loop unrolling to improve memory access pattern
        float val = X[i * INCX];
        // Simplify fminf and fmaxf by using minimum and maximum functions
        X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, val));
    }
}