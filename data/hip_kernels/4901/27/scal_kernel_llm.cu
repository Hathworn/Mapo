#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize thread block indexing for better performance
__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for multiple iterations
    for (int idx = i; idx < N; idx += stride)
    {
        X[idx * INCX] *= ALPHA;
    }
}