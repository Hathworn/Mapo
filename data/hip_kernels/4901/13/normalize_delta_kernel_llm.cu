#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified 1D grid computation
    if (index >= N) return;
    int f = (index / spatial) % filters;

    float std_inv = rsqrtf(variance[f] + .000001f); // Use fast inverse square root
    float val = x[index] - mean[f];
    float delta_val = delta[index];

    delta[index] = delta_val * std_inv +
                   variance_delta[f] * 2.0f * val / (spatial * batch) +
                   mean_delta[f] / (spatial * batch);
}