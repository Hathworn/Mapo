#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified accessing mechanism
    if(id >= n * c) return;

    int b = id / c;
    int k = id % c;

    int out_index = (k + c * b);
    float sum = 0.0f;

    #pragma unroll // Unroll loop for better performance
    for(int i = 0; i < w * h; ++i){
        int in_index = i + h * w * (k + b * c);
        sum += input[in_index];
    }
    output[out_index] = sum / (w * h);
}