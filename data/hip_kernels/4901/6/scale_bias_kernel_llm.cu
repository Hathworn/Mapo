#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    if(offset < size) {
        int filter = blockIdx.y;
        int batch = blockIdx.z;
        // Improve memory access pattern by using register to store repeated index calculation
        int index = (batch * n + filter) * size + offset;
        output[index] *= biases[filter];
    }
}