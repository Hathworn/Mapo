#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;  // Calculate stride for loop unrolling
    
    for (int idx = index; idx < batch * n * n; idx += stride) {
        int f1 = idx / n;
        int f2 = idx % n;
        if (f2 <= f1) continue;  // Skip unnecessary calculations

        float sum = 0;
        float norm1 = 0;
        float norm2 = 0;

        for (int b = 0; b < batch; ++b) {
            int base1 = b * size * n + f1 * size;
            int base2 = b * size * n + f2 * size;
            
            for (int i = 0; i < size; ++i) {
                int i1 = base1 + i;
                int i2 = base2 + i;
                sum += output[i1] * output[i2];
                norm1 += output[i1] * output[i1];
                norm2 += output[i2] * output[i2];
            }
        }
        
        norm1 = rsqrtf(norm1);
        norm2 = rsqrtf(norm2);
        float norm = norm1 * norm2;

        sum = sum * norm;  // Normalize the sum

        for (int b = 0; b < batch; ++b) {
            int base1 = b * size * n + f1 * size;
            int base2 = b * size * n + f2 * size;
            
            for (int i = 0; i < size; ++i) {
                int i1 = base1 + i;
                int i2 = base2 + i;
                atomicAdd(&delta[i1], -scale * sum * output[i2]);
                atomicAdd(&delta[i2], -scale * sum * output[i1]);
            }
        }
    }
}
```
