#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index is within bounds
    if (index < N) { 
        // Precompute constant values outside the loop
        float bias_correction1 = 1.0f - powf(B1, t);
        float bias_correction2 = 1.0f - powf(B2, t);
        float sqrt_bias_correction2 = sqrtf(bias_correction2);

        // Update the parameter using Adam update rule with precomputed values
        float m_hat = m[index] / bias_correction1;
        float v_hat = sqrtf(v[index]) + eps;
        float scaled_rate = rate * sqrt_bias_correction2 / v_hat;

        x[index] -= scaled_rate * m_hat;
    }
}