#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0.0f;

    // Optimize by unrolling the loop over i
    for (int b = 0; b < batch; ++b){
        for (int i = 0; i < size; i += BLOCK){
            int index = p + i + size * (filter + n * b);
            if (p + i < size) {
                // Use coalesced memory access
                sum += delta[index];
            }
        }
    }
    part[p] = sum;
    __syncthreads();

    // Parallel reduction for efficiency
    if (threadIdx.x < BLOCK / 2) {
        part[threadIdx.x] += part[threadIdx.x + BLOCK / 2];
    }
    __syncthreads();

    if (threadIdx.x < BLOCK / 4) {
        part[threadIdx.x] += part[threadIdx.x + BLOCK / 4];
    }
    __syncthreads();

    if (threadIdx.x < BLOCK / 8) {
        part[threadIdx.x] += part[threadIdx.x + BLOCK / 8];
    }
    __syncthreads();

    if (threadIdx.x < BLOCK / 16) {
        part[threadIdx.x] += part[threadIdx.x + BLOCK / 16];
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        for (int i = 0; i < BLOCK / 32; ++i) {
            bias_updates[filter] += part[i];
        }
    }
}