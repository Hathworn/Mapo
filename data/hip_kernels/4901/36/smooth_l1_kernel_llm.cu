#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop over input elements to handle more data than threads
    for (; i < n; i += stride) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff); // Use fast math function

        if(abs_val < 1.0f) {
            error[i] = diff * diff; // Square of the difference for small error
            delta[i] = diff; // Store the difference
        } else {
            error[i] = 2.0f * abs_val - 1.0f; // Adjusted error for large differences
            delta[i] = copysignf(1.0f, diff); // Efficient sign determination
        }
    }
}