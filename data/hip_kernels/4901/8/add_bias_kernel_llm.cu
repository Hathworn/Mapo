#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Ensure offset is within bounds
    if(offset >= size) return;

    // Precompute index for repeated use
    int index = (batch * n + filter) * size + offset;

    // Add bias to output
    output[index] += biases[filter];
}