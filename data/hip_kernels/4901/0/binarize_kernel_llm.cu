#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize boundary check with stride loop
    for(; i < n; i += gridDim.x * blockDim.x) {
        binary[i] = (x[i] >= 0) ? 1 : -1;
    }
}