#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0.0f;

    // Optimize loop ordering for better memory access pattern
    for(int i = p; i < size; i += BLOCK) {
        for(int b = 0; b < batch; ++b) {
            int index = i + size * (filter + n * b);
            sum += delta[index] * x_norm[index];
        }
    }
    
    part[p] = sum;
    __syncthreads();
    
    // Use parallel reduction for performance improvement
    for(int stride = BLOCK / 2; stride > 0; stride >>= 1) {
        if(p < stride) {
            part[p] += part[p + stride];
        }
        __syncthreads();
    }

    // Single thread writes to global memory
    if (p == 0) {
        scale_updates[filter] = part[0];
    }
}