#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    // Process only valid elements
    for (int stride = gridDim.x * blockDim.x; i < N; i += stride) 
    {
        Y[i * INCY] *= X[i * INCX]; // Element-wise multiplication
    }
}