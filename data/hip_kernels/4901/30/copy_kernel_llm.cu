#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use only blockIdx.x for grid
    int stride = gridDim.x * blockDim.x; // Calculate stride for loop
    for (; i < N; i += stride) { // Unroll loop to process multiple elements
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
    }
}