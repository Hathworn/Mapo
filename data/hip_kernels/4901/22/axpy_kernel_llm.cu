#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop through elements that this thread can process
    for (; i < N; i += blockDim.x * gridDim.x) {
        // Perform the AXPY operation
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}