#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    // Use shared memory for faster access and reduce redundant calculations
    __shared__ int shared_minw, shared_minh, shared_minc;
    if (threadIdx.x == 0) {
        shared_minw = minw;
        shared_minh = minh;
        shared_minc = minc;
    }
    __syncthreads();

    // Calculate global id more efficiently
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) return;

    // Unroll the division operations to prevent repeated calculations
    int i = id % shared_minw;
    id /= shared_minw;
    int j = id % shared_minh;
    id /= shared_minh;
    int k = id % shared_minc;
    id /= shared_minc;
    int b = id;

    // Optimize memory access by reducing calculation redundancy
    int out_index = (i + sample * (j + w2 * (k + c2 * b))) * w2;
    int add_index = (i * stride + w1 * (j * stride + h1 * (k + c1 * b))) * w1;
    out[out_index] += add[add_index];
}