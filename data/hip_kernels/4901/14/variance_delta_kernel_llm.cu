#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Calculate the index for this thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return; // Boundary check

    // Initialize variance_delta for the filter to zero
    float v_delta = 0.0f;
    
    // Use shared memory to cache mean and variance for faster access
    __shared__ float shared_mean[1024];
    __shared__ float shared_variance[1024];
    
    if (threadIdx.x < filters) {
        shared_mean[threadIdx.x] = mean[threadIdx.x];
        shared_variance[threadIdx.x] = variance[threadIdx.x];
    }
    __syncthreads();

    // Accumulate variance_delta
    for(int j = 0; j < batch; ++j){
        for(int k = 0; k < spatial; ++k){
            int index = j * filters * spatial + i * spatial + k;
            v_delta += delta[index] * (x[index] - shared_mean[i]);
        }
    }

    // Store the result with optimized variance calculation
    variance_delta[i] = v_delta * (-0.5f) * powf(shared_variance[i] + 1e-6f, -1.5f);
}