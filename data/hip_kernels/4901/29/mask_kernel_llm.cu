#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use blockIdx.z and warp optimization
__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y) * blockDim.x + threadIdx.x;
    if (i < n && mask[i] == mask_num)
    {
        x[i] = mask_num;
    }
}