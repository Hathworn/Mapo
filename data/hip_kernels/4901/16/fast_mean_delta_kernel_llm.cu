#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Parallel reduction for calculating mean delta
    int i, j;
    for (j = 0; j < batch; ++j) {
        for (i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += delta[index];
        }
    }

    __syncthreads();

    // Reduce the result in shared memory
    for (int stride = threads / 2; stride > 0; stride >>= 1) {
        if (id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    // Update mean_delta for the current filter
    if (id == 0) {
        mean_delta[filter] = local[0] * (-1. / sqrt(variance[filter] + .000001f));
    }
}