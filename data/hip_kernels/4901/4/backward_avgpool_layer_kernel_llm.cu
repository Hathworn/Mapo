#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int spatial_size = w * h;  // Calculate once
    int k = id % c;
    int b = id / c;  // Direct division for batch index

    int out_index = k + c * b;
    float delta_val = out_delta[out_index] / spatial_size;  // Compute only once

    int base_index = h * w * (k + b * c);
    for (int i = 0; i < spatial_size; ++i) {
        int in_index = i + base_index;
        in_delta[in_index] += delta_val;
    }
}