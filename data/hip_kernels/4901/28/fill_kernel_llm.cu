#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize for 1D grid and block
    int stride = gridDim.x * blockDim.x; // Calculate global stride
    for (int idx = i; idx < N; idx += stride) // Use stride loop for coalesced access
    {
        X[idx * INCX] = ALPHA;
    }
}