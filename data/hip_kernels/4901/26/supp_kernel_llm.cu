#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate unique global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
  
    // Process elements with a stride loop for better memory access pattern
    for(; i < N; i += stride) {
        if((X[i*INCX] * X[i*INCX]) < (ALPHA * ALPHA)) X[i*INCX] = 0;
    }
}