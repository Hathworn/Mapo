#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Optimized thread index calculation for better readability
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use a loop to process more elements when there are remaining threads
    for (; i < n; i += stride) {
        float b_val = b ? b[i] : 0.0f; // Use ternary once outside of the calculation
        c[i] = s[i] * a[i] + (1.0f - s[i]) * b_val;
    }
}