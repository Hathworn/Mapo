#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x * blockDim.x) + threadIdx.x; // Simplified thread index calculation
    if (s >= size) return;

    float mean = 0;
    #pragma unroll  // Unroll loop for performance
    for(int i = 0; i < n; ++i){
        mean += abs(input[i * size + s]);
    }
    mean /= n;
    #pragma unroll  // Unroll loop for performance
    for(int i = 0; i < n; ++i){
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}