#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    
    // Cache variance calculation
    float inv_var = -1.0f / sqrtf(variance[i] + 0.000001f);
    
    // Accumulate delta values for each filter
    float sum = 0;
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += delta[index];
        }
    }
    
    // Store the mean delta value
    mean_delta[i] = sum * inv_var;
}