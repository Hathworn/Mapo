#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel using shared memory and loop unrolling
__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    extern __shared__ float shared_mean[];  // Shared memory for reduction
    float scale = 1.0f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    shared_mean[threadIdx.x] = 0;  // Initialize shared memory
    __syncthreads();

    for (int j = 0; j < batch; j++) {
        for (int k = threadIdx.x; k < spatial; k += blockDim.x) {  // Unroll loop over spatial
            int index = j * filters * spatial + i * spatial + k;
            shared_mean[threadIdx.x] += x[index];
        }
    }
    
    __syncthreads();

    if (threadIdx.x == 0) {  // Use one thread to accumulate results
        float total_mean = 0;
        for (int t = 0; t < blockDim.x; ++t) {
            total_mean += shared_mean[t];
        }
        mean[i] = total_mean * scale;
    }
}