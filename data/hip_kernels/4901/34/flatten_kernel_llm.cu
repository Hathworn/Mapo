#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation
    if(index >= N) return;

    int spatial_area = layers * spatial; // Pre-compute common multiplication
    int b = index / spatial_area;
    int i = index % spatial_area;
    
    int in_c = i / spatial;
    int in_s = i % spatial;

    int i1 = b * spatial_area + in_c * spatial + in_s;
    int i2 = b * spatial_area + in_s * layers + in_c;

    if (forward) out[i2] = x[i1];
    else out[i1] = x[i2];
}