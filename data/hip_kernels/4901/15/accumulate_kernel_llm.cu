#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    extern __shared__ float shared_sum[]; // Shared memory allocation
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    shared_sum[threadIdx.x] = 0.0f;
    __syncthreads(); // Synchronize to ensure shared memory initialization

    for (int k = threadIdx.x; k < n; k += blockDim.x) {
        shared_sum[threadIdx.x] += x[k * groups + i];
    }
    
    __syncthreads(); // Synchronize the threads for reduction

    // Parallel reduction within block
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Store the result in the output array
    if (threadIdx.x == 0) {
        sum[i] = shared_sum[0];
    }
}