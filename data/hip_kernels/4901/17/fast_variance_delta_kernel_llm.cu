#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    float sum = 0.0f; // Accumulate result locally
    int index;

    // Loop unrolling for better performance
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) { // Stride by threads
            index = j*spatial*filters + filter*spatial + i;
            sum += delta[index] * (x[index] - mean[filter]);
        }
    }

    local[id] = sum;
    __syncthreads(); // Synchronize before reduction

    if (id == 0) {
        float total = 0.0f;
        for (int i = 0; i < threads; ++i) {
            total += local[i];
        }
        variance_delta[filter] = total * -.5f * powf(variance[filter] + .000001f, -1.5f);
    }
}