#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Calculate scale factor
    float scale = 1.0f / (batch * spatial - 1);
    int j, k;
    
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check thread bounds
    if (i >= filters) return;
    
    // Initialize variance accumulator
    float var_acc = 0.0f;
    
    // Loop over batches and spatial dimensions
    for (j = 0; j < batch; ++j) {
        for (k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            
            // Accumulate the squared differences
            float diff = x[index] - mean[i];
            var_acc += diff * diff;
        }
    }
    
    // Apply scale factor
    variance[i] = var_acc * scale;
}