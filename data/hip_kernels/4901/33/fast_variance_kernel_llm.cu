#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0.0f;

    // Unrolling the inner loop for performance 
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            float diff = x[index] - mean[filter];
            local[id] += diff * diff;
        }
    }

    __syncthreads(); // Ensure all threads have written to local

    // Reduce local array to accumulate results
    if (id == 0) {
        float sum = 0.0f;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        variance[filter] = sum / (spatial * batch - 1);
    }
}