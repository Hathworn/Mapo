#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Use shared memory to improve performance by reducing global memory access
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    if (i < n) {
        float diff = truth[i] - pred[i];
        float err = diff * diff;
        delta[i] = diff;
        error[i] = err; // Correct error calculation
        sdata[tid] = err;
    } else {
        sdata[tid] = 0.0f; // Zero out extra thread memory usage
    }

    __syncthreads();

    // Parallel reduction within block (sum the errors)
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write block error sum to global memory
    if (tid == 0) {
        error[blockIdx.x] = sdata[0];
    }
}