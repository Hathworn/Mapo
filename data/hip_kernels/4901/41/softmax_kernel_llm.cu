#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    float sum = 0.0f;
    float largest = -INFINITY;

    // Find the largest element
    for(int i = 0; i < n; ++i) {
        float val = input[i];
        largest = (val > largest) ? val : largest;
    }

    // Compute the exponentials and sum
    for(int i = 0; i < n; ++i) {
        float e = expf((input[i] - largest) / temp);
        sum += e;
        output[i] = e;
    }

    // Normalize the output
    for(int i = 0; i < n; ++i) {
        output[i] /= sum;
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if(b >= batch) return;

    // Launch softmax on batches
    softmax_device(n, input + b * offset, temp, output + b * offset);
}