#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified block index calculation
    if (id < size) {
        float randomValue = rand[id];  // Cache rand[id] in a register to reduce memory access
        input[id] = (randomValue < prob) ? 0 : input[id] * scale;  // Use cached randomValue
    }
}