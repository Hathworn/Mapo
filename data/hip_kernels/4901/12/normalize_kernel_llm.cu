#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    // Calculate the global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;
    
    // Calculate the filter index directly
    int f = (index / spatial) % filters;
    
    // Precompute 1/sqrt(variance[f]) to avoid redundant computation
    float inv_sqrt_var = rsqrtf(variance[f] + .000001f);
    
    // Normalize the input
    x[index] = (x[index] - mean[f]) * inv_sqrt_var;
}