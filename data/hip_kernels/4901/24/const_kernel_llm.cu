#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Optimize thread index calculation using combined block and thread indexing.
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid stride loop to cover all elements.
    while (i < N) {
        X[i * INCX] = ALPHA;
        // Advance by total number of threads.
        i += blockDim.x * gridDim.x;
    }
}