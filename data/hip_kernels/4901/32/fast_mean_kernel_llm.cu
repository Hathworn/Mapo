#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    
    local[id] = 0;

    // Use larger step for coalesced memory access
    for(int j = 0; j < batch; ++j) {
        for(int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += x[index];
        }
    }

    __syncthreads(); // Ensure all threads have completed their work

    // Thread 0 aggregates partial sums
    if (id == 0) {
        float sum = 0;
        for(int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        mean[filter] = sum / (spatial * batch);
    }
}