#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        // Use branchless operation for efficiency
        float val = in[tid];
        out[tid] = val * (val > 0);
    }
}