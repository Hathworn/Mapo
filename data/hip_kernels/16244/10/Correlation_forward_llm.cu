#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Correlation_forward( float *output, int nOutputChannels, int outputHeight, int outputWidth, float *rInput1, int nInputChannels, int inputHeight, int inputWidth, float *rInput2, int pad_size, int kernel_size, int max_displacement, int stride1, int stride2) 
{
    int pInputWidth = inputWidth + 2 * pad_size;
    int pInputHeight = inputHeight + 2 * pad_size;
    
    int kernel_rad = (kernel_size - 1) / 2;
    int displacement_rad = max_displacement / stride2;
    int displacement_size = 2 * displacement_rad + 1;
    
    int n  = blockIdx.x;
    int y1 = blockIdx.y * stride1 + max_displacement + kernel_rad;
    int x1 = blockIdx.z * stride1 + max_displacement + kernel_rad;
    int c = threadIdx.x;
    
    int pdimyxc = pInputHeight * pInputWidth * nInputChannels;
    int pdimxc = pInputWidth * nInputChannels;
    int pdimc = nInputChannels;
    
    int tdimcyx = nOutputChannels * outputHeight * outputWidth;
    int tdimyx = outputHeight * outputWidth;
    int tdimx = outputWidth;
    
    float nelems = kernel_size * kernel_size * pdimc;
    
    __shared__ float prod_sum[THREADS_PER_BLOCK];
    
    // Optimize memory access by reducing shared memory usage
    float sum = 0.0f;
    
    for (int tj = -displacement_rad; tj <= displacement_rad; ++tj) {
        for (int ti = -displacement_rad; ti <= displacement_rad; ++ti) {
            sum = 0.0f;
            int x2 = x1 + ti * stride2;
            int y2 = y1 + tj * stride2;
            
            for (int j = -kernel_rad; j <= kernel_rad; ++j) {
                for (int i = -kernel_rad; i <= kernel_rad; ++i) {
                    for (int ch = c; ch < pdimc; ch += THREADS_PER_BLOCK) {
                        int indx1 = n * pdimyxc + (y1 + j) * pdimxc + (x1 + i) * pdimc + ch;
                        int indx2 = n * pdimyxc + (y2 + j) * pdimxc + (x2 + i) * pdimc + ch;
                        
                        sum += rInput1[indx1] * rInput2[indx2];
                    }
                }
            }
            
            prod_sum[c] = sum;
            __syncthreads();
            
            // Optimize reduction step with loop unrolling
            if (c == 0) {
                float reduce_sum = 0;
                for (int index = 0; index < THREADS_PER_BLOCK; index += 4) {
                    reduce_sum += prod_sum[index] + prod_sum[index + 1] + prod_sum[index + 2] + prod_sum[index + 3];
                }
                int tc = (tj + displacement_rad) * displacement_size + (ti + displacement_rad);
                const int tindx = n * tdimcyx + tc * tdimyx + blockIdx.y * tdimx + blockIdx.z;
                output[tindx] = reduce_sum / nelems;
            }
            __syncthreads();
        }
    }
}