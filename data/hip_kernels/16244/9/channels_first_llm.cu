#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void channels_first(float* input, float* rinput, int channels, int height, int width, int pad_size)
{
    // Optimize calculation by using shared memory and fewer instructions
    int n = blockIdx.x;
    int y = blockIdx.y;
    int x = blockIdx.z;

    int ch_off = threadIdx.x;

    __shared__ float shared_input[THREADS_PER_BLOCK];

    int dimyx = height * width;
    int p_dimx = (width + 2 * pad_size);
    int p_dimy = (height + 2 * pad_size);
    int p_dimyxc = channels * p_dimy * p_dimx;
    int p_dimxc = p_dimx * channels;
    int offset_input = n * channels * dimyx + y * width + x;
    int offset_rinput = n * p_dimyxc + (y + pad_size) * p_dimxc + (x + pad_size) * channels;

    for (int c = ch_off; c < channels; c += THREADS_PER_BLOCK) {
        // Load input data into shared memory for coalescing
        shared_input[threadIdx.x] = input[offset_input + c * dimyx];
        __syncthreads();

        // Write data from shared memory to the output
        rinput[offset_rinput + c] = shared_input[threadIdx.x];
        __syncthreads();
    }
}