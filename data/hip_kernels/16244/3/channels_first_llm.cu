#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void channels_first(float* input, float* rinput, int channels, int height, int width, int pad_size)
{
    // Combined block and thread indexing to improve access pattern
    int n = blockIdx.x;
    int y = blockIdx.y * blockDim.x + threadIdx.x;

    if (y >= height) return; // Avoid accessing out-of-bound elements

    int x = blockIdx.z;
    int dimcyx = channels * height * width;
    int dimyx = height * width;

    int p_dimx = (width + 2 * pad_size);
    int p_dimy = (height + 2 * pad_size);
    int p_dimyxc = channels * p_dimy * p_dimx;
    int p_dimxc = p_dimx * channels;

    for (int c = 0; c < channels; ++c) {
        // Ensuring coalesced memory access for input and rinput
        float value = input[n * dimcyx + c * dimyx + y * width + x];
        rinput[n * p_dimyxc + (y + pad_size) * p_dimxc + (x + pad_size) * channels + c] = value;
    }
}