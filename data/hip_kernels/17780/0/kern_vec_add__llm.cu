#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Implements vector
*/

#ifdef DEBUG
#endif

__global__ void kern_vec_add_(float* x, float* y, float* r, size_t dim)
{
    // Calculate the global thread index
    size_t _i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride for loop only if necessary
    size_t _strd = blockDim.x * gridDim.x;

    // Iterate over data elements
    while (_i < dim) {
        r[_i] = x[_i] + y[_i];
        _i += _strd;
    }
}