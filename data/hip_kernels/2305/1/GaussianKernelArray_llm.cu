#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GaussianKernelArray(const uint8_t *src, uint8_t *dst, int width, int height, int step, int ks)
{
    __shared__ float filter[5][5];
    if (threadIdx.x < ks && threadIdx.y < ks) {
        filter[threadIdx.y][threadIdx.x] = 
            (threadIdx.y == 0 || threadIdx.y == ks-1 || threadIdx.x == 0 || threadIdx.x == ks-1) ? 0.002969017f : 
            (threadIdx.y == 1 || threadIdx.y == ks-2 || threadIdx.x == 1 || threadIdx.x == ks-2) ? 0.01330621f : 
            (threadIdx.y == 2 && threadIdx.x == 2) ? 0.162102822f : 0.098320331f;
    }
    __syncthreads();

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        float sum = 0;
        // Unrolling loop to reduce overhead and improve performance
        for (int dy = 0; dy < ks; ++dy) {
            int offset = (y + dy) * step;
            for (int dx = 0; dx < ks; ++dx) {
                sum += filter[dy][dx] * src[(x + dx) + offset];
            }
        }
        dst[x + y * step] = static_cast<uint8_t>(sum + 0.5f);
    }
}