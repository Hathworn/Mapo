#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReduceHKernelSimple(const uint8_t *src, float *dst, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;  // Calculate stride
    float sum = 0;

    // Unroll the loop and stride across the grid to utilize global memory efficiently.
    for (int y = 0; y < height; y += stride) {
        if (x + y * width < width * height) {
            sum += src[x + y * width];
        }
    }

    // Safely write the sum to the output
    if (x < width) {
        dst[x] = sum;
    }
}