#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TransposeKernelFast(const uint8_t *src, uint8_t *dst, int width, int height)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int xbase = blockIdx.x * blockDim.x;
    int ybase = blockIdx.y * blockDim.y;

    // Use shared memory with minimal padding to avoid bank conflicts
    __shared__ uint8_t sbuf[16][16];

    // Read data from global memory into shared memory
    int x = xbase + tx;
    int y = ybase + ty;
    if (x < width && y < height)
        sbuf[ty][tx] = src[x + y * width];

    __syncthreads();

    // Write transposed data from shared memory to global memory
    x = ybase + tx;
    y = xbase + ty;
    if (x < height && y < width)
        dst[y + x * height] = sbuf[tx][ty];
}