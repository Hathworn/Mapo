#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReduceInitKernel(float *dst, int length)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for better GPU utilization and handle all elements
    for (int idx = x; idx < length; idx += blockDim.x * gridDim.x) {
        dst[idx] = 0.0f;
    }
}