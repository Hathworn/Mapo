#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TransposeKernelFast2(const uint8_t *src, uint8_t *dst, int width, int height)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int xbase = blockIdx.x * 32;
    int ybase = blockIdx.y * 32;

    __shared__ uint8_t sbuf[32][32 + 4];

    // Load data from global memory to shared memory
    {
        int x = xbase + tx;
        int yend = min(ybase + 32, height);
        if (x < width) {
            #pragma unroll  // Unroll the loop to improve performance
            for (int tyy = ty, y = ybase + ty; y < yend; tyy += 8, y += 8) {
                sbuf[tyy][tx] = src[x + y * width];
            }
        }
    }

    __syncthreads();

    // Store data from shared memory to global memory
    {
        int y = ybase + tx;
        int xend = min(xbase + 32, width);
        if (y < height) {
            #pragma unroll  // Unroll the loop to improve performance
            for (int tyy = ty, x = xbase + ty; x < xend; tyy += 8, x += 8) {
                dst[y + x * height] = sbuf[tx][tyy];
            }
        }
    }
}