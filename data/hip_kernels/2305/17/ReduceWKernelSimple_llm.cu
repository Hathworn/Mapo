#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReduceWKernelSimple(const uint8_t *src, float *dst, int width, int height)
{
    int y = blockIdx.x * blockDim.x + threadIdx.x;
    int x = blockIdx.y * 128;

    if (y < height) {
        float sum = 0;
        // Optimize loop to avoid recomputing index each iteration
        int offset = y * width;
        int xend = min(x + 128, width);
        for (int i = x; i < xend; ++i) {
            sum += src[i + offset];
        }
        atomicAdd(&dst[y], sum);
    }
}