#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TransposeKernelShared(const uint8_t *src, uint8_t *dst, int width, int height) 
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int xbase = blockIdx.x * blockDim.x;
    int ybase = blockIdx.y * blockDim.y;

    // Use shared memory buffer for coalesced read and write
    __shared__ uint8_t sbuf[16][17]; // Padding for avoiding bank conflicts

    int x = xbase + tx;
    int y = ybase + ty;

    // Load from global memory to shared memory
    if (x < width && y < height) {
        sbuf[ty][tx] = src[x + y * width];
    }

    __syncthreads();

    // Transpose and store from shared memory to global memory
    x = xbase + ty;
    y = ybase + tx;
    if (x < height && y < width) {
        dst[y + x * height] = sbuf[tx][ty];
    }
}