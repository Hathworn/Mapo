#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReduceHKernelFast(const uint8_t *src, float *dst, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int yInit = blockIdx.y * 128; // Initialize starting y based on blockIdx

    if (x < width) {
        float sum = 0;
        // Optimize: Use shared memory for intermediate sums
        __shared__ float sharedSum[128];

        // Initialize shared memory to zero
        sharedSum[threadIdx.x] = 0;

        // Loop through rows in chunks
        for (int y = yInit; y < height; y += 128) {
            int yend = min(y + 128, height); // Determine end of the current chunk
            for (int i = y; i < yend; ++i) {
                sharedSum[threadIdx.x] += src[x + i * width];
            }
        }

        // Perform a single atomic add after accumulation
        atomicAdd(&dst[x], sharedSum[threadIdx.x]);
    }
}