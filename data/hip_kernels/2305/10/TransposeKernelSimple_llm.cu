#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TransposeKernelSimple(const uint8_t *src, uint8_t *dst, int width, int height)
{
    // Calculate transposed index for coalesced memory access
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Optimize memory access by checking bounds
    if (x < width && y < height)
    {
        int src_index = y * width + x;  // Row-major order calculation
        int dst_index = x * height + y; // Transpose calculation
        dst[dst_index] = src[src_index];
    }
}