#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Sobel_Kernel(unsigned int* pic, int* result, int xsize, int ysize, int thresh)
{
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Early exit threads that are out of bounds
    if (idx >= xsize || idy >= ysize) return;

    // Load shared memory for neighboring pixels (considering block boundaries)
    extern __shared__ unsigned int sharedPic[];
    int sharedIdx = threadIdx.x + 1;
    int sharedIdy = threadIdx.y + 1;
    int sharedWidth = blockDim.x + 2;

    // Load current pixel
    sharedPic[sharedIdx + sharedIdy * sharedWidth] = pic[idx + idy * xsize];

    // Load border pixels into shared memory
    if (threadIdx.x == 0 && idx > 0) 
        sharedPic[sharedIdx - 1 + sharedIdy * sharedWidth] = pic[idx - 1 + idy * xsize];
    if (threadIdx.x == blockDim.x - 1 && idx < xsize - 1) 
        sharedPic[sharedIdx + 1 + sharedIdy * sharedWidth] = pic[idx + 1 + idy * xsize];
    if (threadIdx.y == 0 && idy > 0) 
        sharedPic[sharedIdx + (sharedIdy - 1) * sharedWidth] = pic[idx + (idy - 1) * xsize];
    if (threadIdx.y == blockDim.y - 1 && idy < ysize - 1) 
        sharedPic[sharedIdx + (sharedIdy + 1) * sharedWidth] = pic[idx + (idy + 1) * xsize];

    __syncthreads();

    // Apply Sobel Operator using shared memory
    int gx = sharedPic[sharedIdx - 1 + (sharedIdy - 1) * sharedWidth] - sharedPic[sharedIdx + 1 + (sharedIdy - 1) * sharedWidth] +
             2 * sharedPic[sharedIdx - 1 + sharedIdy * sharedWidth] - 2 * sharedPic[sharedIdx + 1 + sharedIdy * sharedWidth] +
             sharedPic[sharedIdx - 1 + (sharedIdy + 1) * sharedWidth] - sharedPic[sharedIdx + 1 + (sharedIdy + 1) * sharedWidth];

    int gy = sharedPic[sharedIdx - 1 + (sharedIdy - 1) * sharedWidth] + 2 * sharedPic[sharedIdx + (sharedIdy - 1) * sharedWidth] + sharedPic[sharedIdx + 1 + (sharedIdy - 1) * sharedWidth] -
             sharedPic[sharedIdx - 1 + (sharedIdy + 1) * sharedWidth] - 2 * sharedPic[sharedIdx + (sharedIdy + 1) * sharedWidth] - sharedPic[sharedIdx + 1 + (sharedIdy + 1) * sharedWidth];

    // Calculate edge magnitude
    int edgeVal = sqrtf((float)(gx * gx + gy * gy));

    // Apply threshold
    result[idx + idy * xsize] = (edgeVal > thresh) ? 255 : 0;
}