#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to divide accuracy by N using a single thread
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    if (threadIdx.x == 0 && blockIdx.x == 0) { // Ensure single execution
        *accuracy /= N;
    }
}