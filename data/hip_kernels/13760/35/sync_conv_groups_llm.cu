#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups()
{
    // Use __syncthreads() to synchronize threads within the same thread block
    __syncthreads();
}