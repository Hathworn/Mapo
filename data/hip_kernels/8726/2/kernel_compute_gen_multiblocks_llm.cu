#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CELL_PER_THREAD 1

__global__ void kernel_compute_gen_multiblocks(uint8_t *matrix_src, uint8_t *matrix_dst, uint32_t dim_m1, uint32_t cols) {
    // Calculate global thread ID for 1D grid
    uint32_t cell = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute x and y coordinates in matrix from cell index
    uint32_t x1 = cell % cols;
    uint32_t x0 = (x1 + cols - 1) % cols; // Pre-computed modulo operation
    uint32_t x2 = (x1 + 1) % cols;

    uint32_t y1 = (cell - x1);
    uint32_t y0 = (y1 + dim_m1 - cols) % dim_m1; // Pre-computed modulo operation
    uint32_t y2 = (y1 + cols) % dim_m1;

    // Accumulate alive neighbor cells
    uint8_t aliveCells = matrix_src[x0 + y0] + matrix_src[x1 + y0] + matrix_src[x2 + y0] +
                         matrix_src[x0 + y1] + matrix_src[x2 + y1] + 
                         matrix_src[x0 + y2] + matrix_src[x1 + y2] + matrix_src[x2 + y2];

    // Update matrix based on the rules
    matrix_dst[y1 + x1] = (aliveCells == 3 || (aliveCells == 2 && matrix_src[y1 + x1])) ? 1 : 0;
}