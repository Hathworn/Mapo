#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CELL_PER_THREAD 1

__device__ void gpu_swap(void **a, void **b) {
    void *temp = *a;
    *a = *b;
    *b = temp;
}

__global__ void kernel_compute_gen_singleblock(uint8_t *matrix_src, uint8_t *matrix_dst,  uint32_t dim_m1, uint32_t cols, uint32_t iterations, uint32_t cellPerThreads) {
    uint32_t start = threadIdx.x * cellPerThreads;
    uint32_t end =  cellPerThreads + start;

    for (uint32_t iter = 0; iter < iterations; iter++) {
        for (uint32_t cell = start; cell < end; cell++) {
            uint32_t x1 = cell % cols;          // Simplified modulus operation
            uint32_t x0 = (x1 + cols - 1) % cols; // Adjust for negative index
            uint32_t x2 = (x1 + 1) % cols;

            uint32_t y1 = cell - x1;
            uint32_t y0 = (y1 + dim_m1 - cols + 1) & dim_m1; // Utilize & for modulus by power of 2
            uint32_t y2 = (y1 + cols) & dim_m1;

            uint8_t aliveCells = 
                matrix_src[x0 + y0] + matrix_src[x1 + y0] + matrix_src[x2 + y0] +
                matrix_src[x0 + y1] + /* matrix_src[x1 + y1] + */ matrix_src[x2 + y1] +
                matrix_src[x0 + y2] + matrix_src[x1 + y2] + matrix_src[x2 + y2];

            uint8_t currentState = matrix_src[y1 + x1];
            matrix_dst[y1 + x1] = (aliveCells == 3 || (aliveCells == 2 && currentState)) ? 1 : 0;
        }

        __syncthreads(); // Synchronize threads only once per iteration
        gpu_swap((void**)&matrix_dst, (void**)&matrix_src);
    }
}