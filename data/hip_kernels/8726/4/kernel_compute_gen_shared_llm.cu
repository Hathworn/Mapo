#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CELL_PER_THREAD 1

__global__ void kernel_compute_gen_shared(uint8_t *matrix_src, uint8_t *matrix_dst, uint32_t dim_m1, uint32_t cols) {
    extern __shared__ uint8_t shared[];
    uint32_t cell = blockIdx.x * blockDim.x + threadIdx.x;

    uint32_t x1 = cell & (cols - 1);  // Optimized calculation for column index
    uint32_t x0 = (x1 - 1) & (cols - 1);
    uint32_t x2 = (x1 + 1) & (cols - 1);

    uint32_t y1 = cell - x1;
    uint32_t y0 = (y1 - cols) & dim_m1;  // Optimized calculation for row index
    uint32_t y2 = (y1 + cols) & dim_m1;

    // Load matrix data into shared memory with padding
    shared[threadIdx.x + 1] = matrix_src[x1 + y0];
    shared[threadIdx.x + 1 + blockDim.x + 2] = matrix_src[cell];
    shared[threadIdx.x + 1 + (blockDim.x + 2) * 2] = matrix_src[x1 + y2];

    if (threadIdx.x == 0) {
        // Load boundary cells for thread 0
        shared[0] = matrix_src[x0 + y0];
        shared[blockDim.x + 2] = matrix_src[x0 + y1];
        shared[(blockDim.x + 2) * 2] = matrix_src[x0 + y2];
    } else if (threadIdx.x == blockDim.x - 1) {
        // Load boundary cells for last thread
        shared[blockDim.x + 2 - 1] = matrix_src[x2 + y0];
        shared[(blockDim.x + 2) * 2 - 1] = matrix_src[x2 + y1];
        shared[(blockDim.x + 2) * 3 - 1] = matrix_src[x2 + y2];
    }

    __syncthreads();

    uint32_t scell = threadIdx.x + 1 + blockDim.x + 2;

    uint32_t sx1 = scell;  // Index calculations for shared memory
    uint32_t sx0 = (sx1 - 1);
    uint32_t sx2 = (sx1 + 1);

    uint32_t sy1 = scell - sx1;
    uint32_t sy0 = (sy1 - blockDim.x - 2);
    uint32_t sy2 = (sy1 + blockDim.x + 2);

    // Compute the number of alive cells in the neighborhood
    uint8_t aliveCells = shared[sx0 + sy0] + shared[sx1 + sy0] + shared[sx2 + sy0] +
                         shared[sx0 + sy1] + shared[sx2 + sy1] +
                         shared[sx0 + sy2] + shared[sx1 + sy2] + shared[sx2 + sy2];

    // Update the destination matrix based on cell rules
    matrix_dst[y1 + x1] = (aliveCells == 3 || (aliveCells == 2 && shared[scell])) ? 1 : 0;
}