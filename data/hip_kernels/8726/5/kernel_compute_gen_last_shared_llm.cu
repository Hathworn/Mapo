#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CELL_PER_THREAD 1

__global__ void kernel_compute_gen_last_shared(uint8_t *matrix_src, uint8_t *matrix_dst, uint32_t rows, uint32_t cols) {
    __shared__ int shared[3][128 + 2];

    int ix = ((blockDim.x - 2) * blockIdx.x + threadIdx.x) & (cols - 1);
    int iy = (blockIdx.y * blockDim.y + threadIdx.y) & (rows - 1); // Changed to account for block size
    int id = iy * cols + ix;

    int i = threadIdx.y;
    int j = threadIdx.x;

    uint8_t mine = matrix_src[id]; // Keep cell in register
    shared[i][j] = mine;

    __syncthreads();

    if (i == 1 && j > 0 && j < blockDim.x - 1) {
        // Unrolled neighbor computation for fewer memory accesses
        uint8_t aliveCells = shared[i + 1][j] + 
                             shared[i - 1][j] +
                             shared[i][j + 1] +
                             shared[i][j - 1] +
                             shared[i + 1][j + 1] +
                             shared[i - 1][j - 1] +
                             shared[i - 1][j + 1] + 
                             shared[i + 1][j - 1];

        // Direct computation and write back result
        matrix_dst[id] = (aliveCells == 3 || (aliveCells == 2 && mine)) ? 1 : 0;
    }
}