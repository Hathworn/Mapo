#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void assignColIds(int* colIds, const int* colOffsets) {
    int myId = blockIdx.x;
    int start = colOffsets[myId];
    int end = colOffsets[myId + 1];
    int stride = blockDim.x * gridDim.x; // Calculate stride for improved occupancy
    for (int id = start + threadIdx.x; id < end; id += stride) { // Use stride for loop increments
        colIds[id] = myId;
    }
}