#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to divide accuracy by N
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread 0 to perform the operation to avoid race conditions
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}