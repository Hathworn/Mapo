#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Inc1(float *Ad, float *Bd) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx < 1) {
        float a = Ad[tx]; // Store Ad[tx] in a register
        for (int i = 0; i < ITER; ++i) {
            a = a + 1.0f; // Update a instead of Ad[tx]
            float b = a; // Store a in a temporary variable b
            for (int j = 0; j < 256; ++j) {
                Bd[tx] = b; // Use b instead of Ad[tx]
            }
        }
        Ad[tx] = a; // Write back the final result to Ad[tx]
    }
}