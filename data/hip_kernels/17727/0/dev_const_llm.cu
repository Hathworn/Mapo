#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dev_const(float *px, float k) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Check if tid is within array bounds
    if (tid < blockDim.x * gridDim.x) {
        px[tid] = k;
    }
}