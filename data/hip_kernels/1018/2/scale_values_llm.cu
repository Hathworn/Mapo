#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_values(float *num, size_t size, float abs_max)
{
    // Use shared memory for efficient access to thread data
    __shared__ float local_abs_max;
    if (threadIdx.x == 0)
    {
        local_abs_max = abs_max;
    }
    __syncthreads();

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        // Reuse loaded abs_max from shared memory
        num[idx] = (local_abs_max + local_abs_max) * num[idx] - local_abs_max;
    }
}