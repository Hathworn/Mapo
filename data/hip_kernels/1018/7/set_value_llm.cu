#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_value(float value, float *array, unsigned int size) {
    // Calculate global index using blockIdx and threadIdx
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop to handle larger arrays
    for (int i = index; i < size; i += blockDim.x * gridDim.x) {
        array[i] = value;
    }
}