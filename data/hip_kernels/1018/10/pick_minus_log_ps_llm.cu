#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pick_minus_log_ps(float *matrix, float *minus_log_ps, unsigned int *indices, unsigned int row, unsigned int col) {
    // Calculate the index for this thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds before processing
    if (index < row) {
        // Compute offset once and use shared memory for col
        unsigned int idx = index * col + indices[index];
        // Use a local variable to store the matrix value and improve register usage
        float matrix_value = matrix[idx];
        // Perform log computation and update the result in minus_log_ps
        minus_log_ps[index] = -log(matrix_value);
    }
}