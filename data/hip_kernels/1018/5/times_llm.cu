#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void times(float *input, unsigned int input_size, float *output, unsigned int n) {
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Pre-calculate the loop condition
    int total_size = n * input_size;
    
    // Check if index is within bounds
    if (index < total_size) {
        // Calculate input index using modulo
        int input_index = index % input_size;
        
        // Assign input to output using calculated index
        output[index] = input[input_index];
    }
}