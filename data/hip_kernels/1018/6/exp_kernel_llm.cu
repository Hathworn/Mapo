#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void exp_kernel(float *array, unsigned int size) {
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a stride for improved memory access pattern
    int stride = gridDim.x * blockDim.x;
    
    for (int i = index; i < size; i += stride) {
        array[i] = exp(array[i]);
    }
}