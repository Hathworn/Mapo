#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float relu(float a) {
    return a < 0 ? 0 : a;
}

// Improved kernel for relu_derivative
__global__ void relu_derivative(float *upper_grads, float *upper_values, unsigned int upper_size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < upper_size) {
        upper_grads[index] = (upper_values[index] == 0) ? 0.0f : upper_grads[index]; // Use ternary operator for clarity
    }
}