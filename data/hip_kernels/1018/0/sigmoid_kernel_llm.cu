#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float sigmoid(float a) {
    return 1.0 / (1.0 + exp(-a));
}

__global__ void sigmoid_kernel(float *vec, int len) {
    // Using shared memory to improve memory access latency
    extern __shared__ float shared_vec[];

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < len) {
        shared_vec[threadIdx.x] = vec[index];
        __syncthreads(); // Ensure shared memory is fully loaded

        // Compute sigmoid in shared memory
        shared_vec[threadIdx.x] = sigmoid(shared_vec[threadIdx.x]);

        __syncthreads(); // Ensure all computations are done
        vec[index] = shared_vec[threadIdx.x]; // Write back to global memory
    }
}