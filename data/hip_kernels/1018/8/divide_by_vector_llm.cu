#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void divide_by_vector(float *matrix, float *vector, unsigned int row, unsigned int col) {
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (index < row * col) {
        // Precompute vector index only once
        unsigned int vecIndex = index / col; 
        matrix[index] /= vector[vecIndex];
    }
}