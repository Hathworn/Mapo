#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float relu(float a) {
    return a < 0 ? 0 : a;
}
__global__ void relu_kernel(float *vec, int len) {
    // Calculate unique thread index using built-in variables for better readability
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll the loop to optimize memory access and reduce control flow divergence
    for (int i = index; i < len; i += blockDim.x * gridDim.x) {
        vec[i] = relu(vec[i]);
    }
}