#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float sigmoid(float a) {
    return 1.0 / (1.0 + exp (-a));
}

__global__ void sigmoid_derivative(float *upper_grads, float *upper_values, unsigned int upper_size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use warp shuffle to calculate in parallel if possible
    if (index < upper_size) {
        float val = upper_values[index];
        upper_grads[index] *= val * (1.0f - val);
    }
}