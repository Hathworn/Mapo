#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void minus_one(float *matrix, unsigned int *indices, unsigned int row, unsigned int col) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < row) {
        // Load index and col once to reduce redundant calculations
        unsigned int colIndex = indices[index];
        unsigned int matrixIndex = index * col + colIndex;
        // Use atomic operation for better performance if parallel writes may happen
        atomicExch(&matrix[matrixIndex], matrix[matrixIndex] - 1.0f);
    }
}