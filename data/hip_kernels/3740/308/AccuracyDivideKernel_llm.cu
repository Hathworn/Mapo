#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
  // Use single thread to divide, ensuring no race conditions
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    *accuracy /= N;
  }
}