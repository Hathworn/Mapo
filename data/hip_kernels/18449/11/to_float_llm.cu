#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void to_float(float *out, int *in, int size) {
    int element = threadIdx.x + blockDim.x * blockIdx.x;
    // Unroll loop to process elements in chunks for better memory throughput
    for (int i = element; i < size; i += blockDim.x * gridDim.x) {
        out[i] = __int2float_rn(in[i]); // Use intrinsic for integer to float conversion
    }
}