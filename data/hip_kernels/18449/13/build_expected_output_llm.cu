#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void build_expected_output(int *output, int n_rows, int k, const int *labels) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n_rows) {  // Move condition check inside to avoid extra return
        int cur_label = labels[row];
        int base_index = row * k;  // Precompute base index for memory access
        for (int i = 0; i < k; i++) {
            output[base_index + i] = cur_label;  // Use precomputed base index
        }
    }
}