#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_priority(int *new_priority, int n_selected, const int *new_idx, int n_ws, const int *idx, const int *priority) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n_selected) {
        int my_new_idx = new_idx[tid];
        // Use shared memory for idx and priority to optimize memory access
        __shared__ int shared_idx[1024];
        __shared__ int shared_priority[1024];
        
        // Load data from global to shared memory
        for (int i = threadIdx.x; i < n_ws; i += blockDim.x) {
            shared_idx[i] = idx[i];
            shared_priority[i] = priority[i];
        }
        __syncthreads();
        
        // The working set size is limited (~1024 elements) so we just loop through it
        for (int i = 0; i < n_ws; i++) {
            if (shared_idx[i] == my_new_idx) {
                new_priority[tid] = shared_priority[i] + 1;
                break; // Exit early if match is found
            }
        }
    }
}