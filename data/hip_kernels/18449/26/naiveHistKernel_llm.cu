#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void optimizedHistKernel(int* bins, int nbins, int* in, int nrows) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    auto offset = blockIdx.y * nrows;
    auto binOffset = blockIdx.y * nbins;
    
    // Use shared memory to temporarily store bin counts for warp-level atomic operations
    extern __shared__ int sharedBins[];
    int laneId = threadIdx.x % warpSize;

    // Initialize shared memory
    for (int i = threadIdx.x; i < nbins; i += blockDim.x) {
        sharedBins[i] = 0;
    }
    __syncthreads();

    // Process input data
    for (; tid < nrows; tid += stride) {
        int id = in[offset + tid];
        id = max(0, min(id, nbins - 1)); // Clamp id to valid range
        atomicAdd(&sharedBins[id], 1);
    }
    __syncthreads();

    // Aggregate results from shared memory to global memory
    for (int i = laneId; i < nbins; i += warpSize) {
        atomicAdd(bins + binOffset + i, sharedBins[i]);
    }
}