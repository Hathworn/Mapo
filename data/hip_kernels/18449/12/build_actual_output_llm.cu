#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void build_actual_output(int *output, int n_rows, int k, const int *idx_labels, const int64_t *indices) {
    int element = threadIdx.x + blockDim.x * blockIdx.x;
    int total_elements = n_rows * k;
  
    // Use grid-stride loop for better utilization of all threads
    for (; element < total_elements; element += blockDim.x * gridDim.x) {
        int ind = (int)indices[element];
        output[element] = idx_labels[ind];
    }
}