#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_unavailable(bool *available, int n_rows, const int *idx, int n_selected) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Improve memory access by using shared memory
    __shared__ int shared_idx[256]; 

    if (tid < n_selected) {
        // Load indexes into shared memory
        shared_idx[threadIdx.x] = idx[tid];
        __syncthreads(); // Ensure all writes to shared memory are complete

        // Use shared memory for updating availability
        atomicExch(&available[shared_idx[threadIdx.x]], false);
    }
}