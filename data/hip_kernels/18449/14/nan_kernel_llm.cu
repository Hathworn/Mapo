#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nan_kernel(float* data, const bool* mask, int len, float nan) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Use shared memory to reduce memory access latency
    __shared__ bool shared_mask[BLOCK_SIZE];
    if (tid < len) {
        shared_mask[threadIdx.x] = mask[tid];
        __syncthreads();
        if (!shared_mask[threadIdx.x]) data[tid] = nan;
    }
}
```
