#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void postprocess_matrix(float* matrix, long* long_indices, int* indices, unsigned int N_POINTS, unsigned int K)
{
    // Calculate unique thread ID
    int TID = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Exit if out of bounds
    if (TID >= N_POINTS * K) return;

    // Transfer long index to integer index
    indices[TID] = static_cast<int>(long_indices[TID]);
}