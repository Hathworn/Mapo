#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void timedReduction(const float *input, float *output, clock_t *timer)
{
    extern __shared__ float shared[];

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int gridSize = blockDim.x * 2;

    if (tid == 0) timer[bid] = clock(); // Start timer

    // Copy input.
    shared[tid] = input[tid + bid * gridSize];
    shared[tid + blockDim.x] = input[tid + blockDim.x + bid * gridSize];
  
    // Perform reduction to find minimum.
    for (int stride = blockDim.x; stride > 0; stride >>= 1) 
    {
        __syncthreads(); // Ensure all threads have updated shared memory

        if (tid < stride)
        {
            float f0 = shared[tid];
            float f1 = shared[tid + stride];

            // Assign minimum value
            shared[tid] = f1 < f0 ? f1 : f0;
        }
    }

    // Write result.
    if (tid == 0) output[bid] = shared[0]; // Store minimum in output

    __syncthreads();

    if (tid == 0) timer[bid + gridDim.x] = clock(); // End timer
}