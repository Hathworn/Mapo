#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void do_sum_merge(int *datas, int n) {
    // Calculate 1D thread ID
    int tid = blockDim.x * threadIdx.y + threadIdx.x;

    // Loop reduction
    while (n > 1) {
        int pair_index = n - 1 - tid;
        if (tid < (n + 1) / 2 && pair_index != tid) {
            datas[tid] += datas[pair_index];
            printf("%d->%d->%d\n", n, tid, datas[tid]);
        }
        // Divide n by 2 for next reduction step
        n /= 2;
        __syncthreads(); // Synchronize threads to ensure completion of current step
    }
}