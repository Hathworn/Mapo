#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

// Optimization in the merge_sort kernel function
__global__ void merge_sort(int *datas, int n) {
    int tid = blockDim.x * threadIdx.y + threadIdx.x;
    extern __shared__ int shared[];

    // Load data into shared memory
    if (tid < n) shared[tid] = datas[tid];
    __syncthreads();

    // Merge sort logic optimized
    for (int gap = 2, cnt = 1; gap < n * 2; gap <<= 1, cnt++) {
        if (tid % gap == 0) {
            int left = tid + n * ((cnt + 1) % 2);
            int mid = tid + gap / 2 + n * ((cnt + 1) % 2);
            int end = tid + gap + ((cnt + 1) % 2) * n;
            int full_end = (1 + (cnt + 1) % 2) * n;
            int res_ind = n * (cnt % 2) + tid;

            // Efficient merging process
            // Use shared memory efficiently
            while ((left < mid && left < full_end) || (right < end && right < full_end)) {
                if (!(left < mid && left < full_end)) {
                    shared[res_ind] = shared[right];
                    right++;
                } else if (!(right < end && right < full_end)) {
                    shared[res_ind] = shared[left];
                    left++;
                } else {
                    shared[res_ind] = (shared[right] > shared[left]) ? shared[left++] : shared[right++];
                }
                res_ind++;
            }
        }
        __syncthreads();
    }
    
    // Update global memory
    if (tid < n) datas[tid] = shared[tid + ((cnt + 1) % 2) * n];
}