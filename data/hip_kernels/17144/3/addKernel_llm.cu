#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b) {
    int i = blockDim.x * blockIdx.x + threadIdx.x; // Use blockIdx and blockDim for larger data
    c[i] = a[i] + b[i];  // Compute addition
}