#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_gkylCartFieldAccumulateOffset(unsigned sInp, unsigned sOut, unsigned nCells, unsigned compStart, unsigned nCompInp, unsigned nCompOut, double fact, const double *inp, double *out) {
    // Calculate the global thread index
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < nCells) { // Check boundary condition for thread index
        if (nCompInp < nCompOut) {
            for (unsigned c = 0; c < nCompInp; ++c) {
                // Directly calculate index to avoid repeated computation
                unsigned outIdx = sOut + i * nCompOut + compStart + c;
                unsigned inpIdx = sInp + i * nCompInp + c;
                out[outIdx] += fact * inp[inpIdx];
            }
        } else {
            for (unsigned c = 0; c < nCompOut; ++c) {
                // Directly calculate index to avoid repeated computation
                unsigned outIdx = sOut + i * nCompOut + c;
                unsigned inpIdx = sInp + i * nCompInp + compStart + c;
                out[outIdx] += fact * inp[inpIdx];
            }
        }
    }
}