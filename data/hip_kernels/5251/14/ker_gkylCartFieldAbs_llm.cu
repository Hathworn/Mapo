#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_gkylCartFieldAbs(unsigned s, unsigned nv, double *out)
{
    // Calculate unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x + s;
    
    // Optimize by unrolling loop to maximize memory access coherency and reduce overhead
    int stride = blockDim.x * gridDim.x;
    for (int n = idx; n < s + nv; n += stride) {
        out[n] = fabs(out[n]);
    }
}