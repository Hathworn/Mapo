#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_gkylCartFieldScale(unsigned s, unsigned nv, double fact, double *out)
{
    // Calculate the global index offset
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Reduce number of iterations by unrolling the loop
    for (int i = n; i < nv; i += blockDim.x * gridDim.x)
    {
        int idx = s + i;
        if (idx < s + nv)
            out[idx] *= fact;
    }
}