#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_gkylCartFieldAssignAll(unsigned s, unsigned nv, double val, double *out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + s;
    int stride = blockDim.x * gridDim.x; 
    // Calculate global index and loop stride

    for (int n = idx; n < s + nv; n += stride)
    {
        // Assign value only within the range of s to s + nv
        out[n] = val;
    }
}