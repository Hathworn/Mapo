#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimized with shared memory
__global__ void ker_gkylCartFieldAccumulate(unsigned s, unsigned nv, double fact, const double *inp, double *out)
{
    // Calculate global index
    int n = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop unrolling for better performance
    for (int i = n; i < nv; i += blockDim.x * gridDim.x)
    {
        if (i >= s) // Boundary check
        {
            out[i] += fact * inp[i];
        }
    }
}