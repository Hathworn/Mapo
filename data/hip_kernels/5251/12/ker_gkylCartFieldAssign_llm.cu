#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_gkylCartFieldAssign(unsigned s, unsigned nv, double fact, const double *inp, double *out)
{
    // Calculate global thread index
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    // Adjust starting point by s
    int start = n + s;
    // Cache stride once
    int stride = blockDim.x * gridDim.x;

    // Loop over range using calculated stride
    for (int i = start; i < s + nv; i += stride) {
        // Update output
        out[i] = fact * inp[i];
    }
}