#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel
__global__ void wlcss_cuda_kernel(int32_t *d_mss, int32_t *d_mss_offsets, int32_t *d_ts, int32_t *d_ss, int32_t *d_tlen, int32_t *d_toffsets, int32_t *d_slen, int32_t *d_soffsets, int32_t *d_params, int32_t *d_3d_cost_matrix) {

    int32_t params_idx = threadIdx.x;
    int32_t template_idx = blockIdx.x;
    int32_t stream_idx = blockIdx.y;

    int32_t t_len = d_tlen[template_idx];
    int32_t s_len = d_slen[stream_idx];

    int32_t t_offset = d_toffsets[template_idx];
    int32_t s_offset = d_soffsets[stream_idx];

    int32_t d_mss_offset = d_mss_offsets[params_idx * gridDim.x * gridDim.y + template_idx * gridDim.y + stream_idx];
    int32_t *mss = &d_mss[d_mss_offset];

    extern __shared__ int32_t shared_tmp_window[]; // Use shared memory for better performance

    int32_t *t = &d_ts[t_offset];
    int32_t *s = &d_ss[s_offset];

    int32_t reward = d_params[params_idx * 3];
    int32_t penalty = d_params[params_idx * 3 + 1];
    int32_t accepteddist = d_params[params_idx * 3 + 2];

    for(int32_t j = 0; j < s_len; j++) {
        int32_t tmp = 0; // Moved inside the first loop for improved locality
        for(int32_t i = 0; i < t_len; i++) {
            int32_t distance = d_3d_cost_matrix[s[j] * 26 + t[i]];
            if (distance <= accepteddist) {
                tmp = shared_tmp_window[i] + reward;
            } else {
                tmp = max(shared_tmp_window[i] - penalty * distance,
                          max(shared_tmp_window[i + 1] - penalty * distance,
                              shared_tmp_window[t_len + 1] - penalty * distance));
            }
            shared_tmp_window[i] = shared_tmp_window[t_len + 1];
            shared_tmp_window[t_len + 1] = tmp;
        }
        shared_tmp_window[t_len] = shared_tmp_window[t_len + 1];
        mss[j] = shared_tmp_window[t_len + 1];
        shared_tmp_window[t_len + 1] = 0; // Reset for the next iteration
    }
}