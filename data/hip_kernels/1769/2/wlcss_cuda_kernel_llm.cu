#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void wlcss_cuda_kernel(int32_t *d_mss, int32_t *d_mss_offsets, int32_t *d_ts, int32_t *d_ss, int32_t *d_tlen, int32_t *d_toffsets, int32_t *d_slen, int32_t *d_soffsets, int32_t *d_params, int32_t *d_tmp_windows, int32_t *d_tmp_windows_offsets) {

    int32_t params_idx = threadIdx.x;
    int32_t template_idx = threadIdx.x;
    int32_t stream_idx = blockIdx.x;

    int32_t t_len = d_tlen[template_idx];
    int32_t s_len = d_slen[stream_idx];

    int32_t t_offset = d_toffsets[template_idx];
    int32_t s_offset = d_soffsets[stream_idx];

    int32_t d_mss_offset = d_mss_offsets[stream_idx * blockDim.x + template_idx];
    int32_t d_tmp_windows_offset = d_tmp_windows_offsets[stream_idx * blockDim.x + template_idx];

    int32_t *tmp_window = &d_tmp_windows[d_tmp_windows_offset];
    int32_t *mss = &d_mss[d_mss_offset];

    int32_t *t = &d_ts[t_offset];
    int32_t *s = &d_ss[s_offset];

    int32_t reward = d_params[params_idx * 3];
    int32_t penalty = d_params[params_idx * 3 + 1];
    int32_t accepteddist = d_params[params_idx * 3 + 2];

    // Cache t_len+1 for repeated use
    int32_t t_len_plus_1 = t_len + 1;

    for (int32_t j = 0; j < s_len; j++) {
        // Use register variable to reduce memory accesses
        int32_t last_tmp = tmp_window[t_len_plus_1];

        for (int32_t i = 0; i < t_len; i++) {
            int32_t distance = abs(s[j] - t[i]);

            if (distance <= accepteddist) {
                last_tmp = tmp_window[i] + reward;
            } else {
                last_tmp = max(tmp_window[i] - penalty * distance,
                               max(tmp_window[i + 1] - penalty * distance,
                                   last_tmp - penalty * distance));
            }

            tmp_window[i] = tmp_window[t_len_plus_1];
            tmp_window[t_len_plus_1] = last_tmp;
        }

        tmp_window[t_len] = tmp_window[t_len_plus_1];
        mss[j] = tmp_window[t_len_plus_1];
        tmp_window[t_len_plus_1] = 0; // Reset for next iteration
    }
}