#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void wlcss_cuda_kernel(int32_t *d_mss, int32_t *d_mss_offsets, int32_t *d_ts, int32_t *d_ss, int32_t *d_tlen, int32_t *d_toffsets, int32_t *d_slen, int32_t *d_soffsets, int32_t *d_params, int32_t *d_tmp_windows, int32_t *d_tmp_windows_offsets, int32_t *d_2d_cost_matrix) {

    int32_t params_idx = threadIdx.x;
    int32_t template_idx = blockIdx.x;
    int32_t stream_idx = blockIdx.y;

    int32_t t_len = d_tlen[template_idx];
    int32_t s_len = d_slen[stream_idx];

    int32_t t_offset = d_toffsets[template_idx];
    int32_t s_offset = d_soffsets[stream_idx];

    int32_t d_mss_offset = d_mss_offsets[params_idx * gridDim.x * gridDim.y + template_idx * gridDim.y + stream_idx];
    int32_t d_tmp_windows_offset = d_tmp_windows_offsets[params_idx * gridDim.x * gridDim.y + template_idx * gridDim.y + stream_idx];

    int32_t *tmp_window = &d_tmp_windows[d_tmp_windows_offset];
    int32_t *mss = &d_mss[d_mss_offset];

    int32_t *t = &d_ts[t_offset];
    int32_t *s = &d_ss[s_offset];

    int32_t reward = d_params[params_idx * 3];
    int32_t penalty = d_params[params_idx * 3 + 1];
    int32_t accepteddist = d_params[params_idx * 3 + 2];

    for (int32_t j = 0; j < s_len; j++) {
        int32_t prev_tmp = 0; // Temporary variable to hold the previous tmp value

        // Loop unrolling for optimization
        for (int32_t i = 0; i < t_len; i += 2) {
            int32_t distance1 = d_2d_cost_matrix[s[j] * 8 + t[i]];
            int32_t distance2 = (i + 1 < t_len) ? d_2d_cost_matrix[s[j] * 8 + t[i + 1]] : INT_MAX;

            // First element update
            int32_t tmp1 = (distance1 <= accepteddist) ? (tmp_window[i] + reward) :
               max(tmp_window[i] - penalty * distance1, max(tmp_window[i + 1] - penalty * distance1, prev_tmp - penalty * distance1));
            prev_tmp = tmp_window[i]; // Update prev_tmp after using it
            tmp_window[i] = prev_tmp;

            // Second element update (if within bounds)
            if (i + 1 < t_len) {
                int32_t tmp2 = (distance2 <= accepteddist) ? (tmp_window[i + 1] + reward) :
                   max(tmp_window[i + 1] - penalty * distance2, max(tmp_window[i + 2] - penalty * distance2, tmp1 - penalty * distance2));
                tmp_window[i + 1] = tmp2;
                prev_tmp = tmp1;
            }
        }

        mss[j] = tmp_window[t_len];
        tmp_window[t_len] = 0;
    }
}