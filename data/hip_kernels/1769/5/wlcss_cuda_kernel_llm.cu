#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Define optimized kernel
__global__ void wlcss_cuda_kernel(int32_t *d_mss, int32_t *d_mss_offsets, int32_t *d_ts, int32_t *d_ss, int32_t *d_tlen, int32_t *d_toffsets, int32_t *d_slen, int32_t *d_soffsets, int32_t *d_params, int32_t *d_tmp_windows, int32_t *d_tmp_windows_offsets){

    // Get indices and parameters
    int32_t params_idx = threadIdx.x;
    int32_t template_idx = blockIdx.x;
    int32_t stream_idx = blockIdx.y;

    int32_t t_len = d_tlen[template_idx];
    int32_t s_len = d_slen[stream_idx];
    
    int32_t t_offset = d_toffsets[template_idx];
    int32_t s_offset = d_soffsets[stream_idx];

    int32_t d_mss_offset = d_mss_offsets[params_idx * gridDim.x * gridDim.y + template_idx * gridDim.y + stream_idx];
    int32_t d_tmp_windows_offset = d_tmp_windows_offsets[params_idx * gridDim.x * gridDim.y + template_idx * gridDim.y + stream_idx];

    int32_t *tmp_window = &d_tmp_windows[d_tmp_windows_offset];
    int32_t *mss = &d_mss[d_mss_offset];

    int32_t *t = &d_ts[t_offset];
    int32_t *s = &d_ss[s_offset];

    int32_t reward = d_params[params_idx * 3];
    int32_t penalty = d_params[params_idx * 3 + 1];
    int32_t accepteddist = d_params[params_idx * 3 + 2];

    // Initialize temporary variables
    int32_t tmp = 0;
    int32_t prev_tmp = 0;

    // Iterate over streams and templates
    for (int32_t j = 0; j < s_len; j++) {
        for (int32_t i = 0; i <= t_len; i++) {
            // Compute distance and update tmp
            int32_t distance = (i < t_len) ? abs(s[j] - t[i]) : 0;
            
            if (i < t_len && distance <= accepteddist) {
                tmp = tmp_window[i] + reward;
            } else {
                int32_t penalty_val = penalty * distance;
                tmp = max(prev_tmp - penalty_val, max(tmp_window[i] - penalty_val, tmp_window[t_len] - penalty_val));
            }
            
            // Swap update values efficently
            prev_tmp = tmp_window[i];
            tmp_window[i] = tmp;
        }

        // Store outcome and reset temporary window value
        mss[j] = tmp;
        tmp_window[t_len] = 0;
    }
}
```
