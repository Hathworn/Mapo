#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void wlcss_cuda_kernel(int32_t *d_mss, int32_t *d_mss_offsets, int32_t *d_ts, int32_t *d_ss, int32_t *d_tlen, int32_t *d_toffsets, int32_t *d_slen, int32_t *d_soffsets, int32_t *d_params, int32_t *d_tmp_windows, int32_t *d_tmp_windows_offsets, int32_t *d_3d_cost_matrix){

    int32_t params_idx = threadIdx.x;
    int32_t template_idx = blockIdx.x;
    int32_t stream_idx = blockIdx.y;

    int32_t t_len = d_tlen[template_idx];
    int32_t s_len = d_slen[stream_idx];

    int32_t t_offset = d_toffsets[template_idx];
    int32_t s_offset = d_soffsets[stream_idx];

    int32_t d_mss_offset = d_mss_offsets[params_idx*gridDim.x*gridDim.y+template_idx*gridDim.y+stream_idx];
    int32_t d_tmp_windows_offset = d_tmp_windows_offsets[params_idx*gridDim.x*gridDim.y+template_idx*gridDim.y+stream_idx];

    int32_t *tmp_window = &d_tmp_windows[d_tmp_windows_offset];
    int32_t *mss = &d_mss[d_mss_offset];

    int32_t *t = &d_ts[t_offset];
    int32_t *s = &d_ss[s_offset];

    int32_t reward = d_params[params_idx*3];
    int32_t penalty = d_params[params_idx*3+1];
    int32_t accepteddist = d_params[params_idx*3+2];

    // Variables moved outside the loop
    int32_t tmp = 0;
    int32_t tmp_window_next;
    int32_t t_val, s_val, distance;

    // Use shared memory to reduce global memory access
    __shared__ int32_t shared_tmp_window[1024]; // Assuming max t_len is 1024 for illustration
    
    for(int32_t j = 0; j < s_len; j++){
        s_val = s[j];

        // Load initial part of tmp_window into shared memory
        for (int i = threadIdx.x; i <= t_len; i += blockDim.x) {
            shared_tmp_window[i] = tmp_window[i];
        }
        __syncthreads();

        for(int32_t i = 0; i < t_len; i++){
            t_val = t[i];
            distance = d_3d_cost_matrix[s_val * 26 + t_val];
            
            // Perform calculations using shared memory
            tmp_window_next = (distance <= accepteddist) ? 
                shared_tmp_window[i] + reward : 
                max(shared_tmp_window[i] - penalty * distance,
                    max(shared_tmp_window[i + 1] - penalty * distance, 
                    shared_tmp_window[t_len + 1] - penalty * distance));
            
            __syncthreads();
            shared_tmp_window[i] = shared_tmp_window[t_len + 1];
            shared_tmp_window[t_len + 1] = tmp_window_next;
            __syncthreads();
        }

        __syncthreads();
        // Synchronize and write results back to global memory
        if (threadIdx.x == 0) {
            shared_tmp_window[t_len] = shared_tmp_window[t_len + 1];
            mss[j] = shared_tmp_window[t_len + 1];
            shared_tmp_window[t_len + 1] = 0;
        }
        __syncthreads();
        
        for (int i = threadIdx.x; i <= t_len; i += blockDim.x) {
            tmp_window[i] = shared_tmp_window[i];
        }
        __syncthreads();
    }
}