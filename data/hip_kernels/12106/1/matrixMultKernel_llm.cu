#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16 // Define block size

__global__ void matrixMultKernel(float *d_A, float *d_B, float *d_C, int N) {
    // Shared memory for sub-matrices
    __shared__ float A_shared[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float B_shared[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    float Cvalue = 0;

    for (int m = 0; m < N / BLOCK_SIZE; ++m) {
        // Load data into shared memory
        if (row < N && (m * BLOCK_SIZE + threadIdx.x) < N) {
            A_shared[threadIdx.y][threadIdx.x] = d_A[row * N + m * BLOCK_SIZE + threadIdx.x];
        } else {
            A_shared[threadIdx.y][threadIdx.x] = 0.0;
        }
        if (col < N && (m * BLOCK_SIZE + threadIdx.y) < N) {
            B_shared[threadIdx.y][threadIdx.x] = d_B[(m * BLOCK_SIZE + threadIdx.y) * N + col];
        } else {
            B_shared[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads();

        // Compute partial dot product
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Cvalue += A_shared[threadIdx.y][k] * B_shared[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < N && col < N) {
        d_C[row * N + col] = Cvalue; // Write back the computed value
    }
}