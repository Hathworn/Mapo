#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void merge_sum(unsigned char * img_all, unsigned char * img, float * selection, float * selection_sum, int n, int stride)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    // Early exit if out-of-bounds
    if (x >= width || y >= width) return;

    int idx;
    float weight = 0;

    // Use shared memory for intermediate image data
    __shared__ unsigned char shared_img[TILE_DIM * BLOCK_ROWS * 3];

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {

        // Initialize shared memory
        shared_img[((j + threadIdx.y) * TILE_DIM + threadIdx.x) * 3] = 0;
        shared_img[((j + threadIdx.y) * TILE_DIM + threadIdx.x) * 3 + 1] = 0;
        shared_img[((j + threadIdx.y) * TILE_DIM + threadIdx.x) * 3 + 2] = 0;

        for (idx = 0; idx < n; idx++) {
            weight = selection[idx * stride + ((y + j) * width + x)] / selection_sum[((y + j) * width + x)];

            int offset = idx * stride * 3 + 3 * ((y + j) * width + x);
            
            // Accumulate weighted values in shared memory
            shared_img[((j + threadIdx.y) * TILE_DIM + threadIdx.x) * 3] += (unsigned char)(img_all[offset] * weight);
            shared_img[((j + threadIdx.y) * TILE_DIM + threadIdx.x) * 3 + 1] += (unsigned char)(img_all[offset + 1] * weight);
            shared_img[((j + threadIdx.y) * TILE_DIM + threadIdx.x) * 3 + 2] += (unsigned char)(img_all[offset + 2] * weight);
        }

        // Sync threads before writing back to global memory
        __syncthreads();

        // Write back results from shared memory
        img[3 * ((y + j) * width + x)] = shared_img[((j + threadIdx.y) * TILE_DIM + threadIdx.x) * 3];
        img[3 * ((y + j) * width + x) + 1] = shared_img[((j + threadIdx.y) * TILE_DIM + threadIdx.x) * 3 + 1];
        img[3 * ((y + j) * width + x) + 2] = shared_img[((j + threadIdx.y) * TILE_DIM + threadIdx.x) * 3 + 2];
    }
}