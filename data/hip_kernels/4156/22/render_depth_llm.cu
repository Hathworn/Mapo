#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void render_depth(float *points3d_polar, unsigned int * depth_render)
{
    // Calculate global thread positions
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int w = gridDim.x * TILE_DIM;
    int h = w / 2;

    // Declare shared memory for coalesced memory access and minimize global memory latency
    __shared__ float shared_points3d_polar[TILE_DIM * BLOCK_ROWS * 3];

    // Load data into shared memory
    int local_index = threadIdx.y * TILE_DIM + threadIdx.x;
    int global_index = y * w + x;
    if (y < h && x < w) {
        shared_points3d_polar[local_index * 3 + 0] = points3d_polar[global_index * 3 + 0];
        shared_points3d_polar[local_index * 3 + 1] = points3d_polar[global_index * 3 + 1];
        shared_points3d_polar[local_index * 3 + 2] = points3d_polar[global_index * 3 + 2];
    }
    __syncthreads();

    // Process each row block in the TILE_DIM step
    for (int j = threadIdx.y; j < TILE_DIM; j += BLOCK_ROWS)
    {
        int iw = x;
        int ih = y + j;
        if (ih < h && iw < w) {
            // Use loaded shared memory
            float theta = shared_points3d_polar[(j * TILE_DIM + threadIdx.x) * 3 + 1];
            float phi = shared_points3d_polar[(j * TILE_DIM + threadIdx.x) * 3 + 2];
            float radius = shared_points3d_polar[(j * TILE_DIM + threadIdx.x) * 3 + 0];

            int tx = round((theta + M_PI) / (2 * M_PI) * w - 0.5);
            int ty = round(phi / M_PI * h - 0.5);
            int this_depth = (int)(512 * radius);

            // Use atomic operation to update the minimum depth
            if (ty < h && tx < w) {
                atomicMin(&depth_render[(ty * w + tx)], this_depth);
            }
        }
    }
}
```
