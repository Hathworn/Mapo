#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void to3d_point(float *depth, float *points3d)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int w = gridDim.x * TILE_DIM;
    int h = w / 2;

    // Unroll the loop to increase parallel execution efficiency
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
        int iw = x;
        int ih = y + j;
        
        // Reduce global memory access
        int index = ih * w + iw;
        float depth_point = depth[index] * 128.0f;
        float phi = ((float)(ih) + 0.5f) / (float)h * M_PI;
        float theta = ((float)(iw) + 0.5f) / (float)w * 2 * M_PI + M_PI;

        // Precalculate common expressions
        float sin_phi = sin(phi);
        float cos_phi = cos(phi);
        float cos_theta = cos(theta);
        float sin_theta = sin(theta);

        points3d[index * 4 + 0] = depth_point * sin_phi * cos_theta;
        points3d[index * 4 + 1] = depth_point * sin_phi * sin_theta;
        points3d[index * 4 + 2] = depth_point * cos_phi;
        points3d[index * 4 + 3] = 1.0f;
    }
}