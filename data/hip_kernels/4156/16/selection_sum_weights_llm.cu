#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void selection_sum_weights(float * selection_sum, float * selection, int n, int stride) {
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;
    float local_sum = 0.0f;
    
    // Loop unrolling for efficient memory access and computation
    #pragma unroll
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        local_sum = 0.0f;
        for (int idx = 0; idx < n; idx++) {
            local_sum += selection[idx * stride + ((y + j) * width + x)];
        }
        atomicAdd(&(selection_sum[(y + j) * width + x]), local_sum);
    }
}