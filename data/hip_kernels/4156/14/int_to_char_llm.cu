#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void int_to_char(int * img2, unsigned char * img)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    // Optimize loop by unrolling for efficiency
    #pragma unroll
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        int idx = (y + j) * width + x;
        int val = img2[idx];
        
        img[3 * idx    ] = val / (256 * 256);
        img[3 * idx + 1] = (val / 256) % 256;
        img[3 * idx + 2] = val % 256;
    }
}