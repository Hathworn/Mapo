#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void char_to_int(int * img2, unsigned char * img)
{
    // Calculate the starting x and y index for the thread
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;

    // Calculate image width
    int width = gridDim.x * TILE_DIM;

    // Optimize: Unroll the loop to improve memory access pattern and reduce loop overhead
    #pragma unroll
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        int index = (y + j) * width + x;
        int r = img[3 * index];
        int g = img[3 * index + 1];
        int b = img[3 * index + 2];

        // Convert RGB to int and store in img2
        img2[index] = (r << 16) | (g << 8) | b;
    }
}