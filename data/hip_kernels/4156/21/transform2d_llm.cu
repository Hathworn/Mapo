#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transform2d(float *points3d_after, float fov_scale)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int w = gridDim.x * TILE_DIM;

    // Utilize shared memory for better memory access efficiency
    __shared__ float shared_points[TILE_DIM * BLOCK_ROWS * 3];
    int local_tid = threadIdx.y * TILE_DIM + threadIdx.x;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
        int iw = x;
        int ih = y + j;
        int global_idx = (ih * w + iw) * 3;

        // Load points into shared memory
        shared_points[local_tid * 3 + 0] = points3d_after[global_idx + 0];
        shared_points[local_tid * 3 + 1] = points3d_after[global_idx + 1];
        shared_points[local_tid * 3 + 2] = points3d_after[global_idx + 2];
        __syncthreads();

        float x = shared_points[local_tid * 3 + 0];
        float y = shared_points[local_tid * 3 + 1];
        float z = shared_points[local_tid * 3 + 2];

        float x2 = fov_scale * x;

        // Use branchless programming for more efficient conditional updates
        bool condition = (x2 > 0) && (y < x2 * 1.1) && (y > -x2 * 1.1) && (z < x2 * 1.1) && (z > -x2 * 1.1);
        float denom = x2 + 1e-5;

        points3d_after[global_idx + 0] = x; // Unchanged
        points3d_after[global_idx + 1] = condition ? y / denom : -1;
        points3d_after[global_idx + 2] = condition ? -z / denom : -1;
    }
}