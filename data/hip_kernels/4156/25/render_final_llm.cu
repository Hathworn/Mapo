#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void render_final(float *points3d_polar, float *selection, float *depth_render, int *img, int *render, int oh, int ow) {
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int w = gridDim.x * TILE_DIM;
    int h = w / 2;
    int maxsize = oh * ow;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        int iw = x;
        int ih = y + j;

        // Precompute index offsets
        int index_current = (ih * w + iw) * 3;
        int index_next_w = (ih * w + iw + 1) * 3;
        int index_next_h = ((ih + 1) * w + iw) * 3;
        int index_next_wh = ((ih + 1) * w + iw + 1) * 3;

        // Precompute repeated calculations
        float half_ow = 0.5 * ow;
        float half_oh = 0.5 * oh;

        float tx_offset = ((points3d_polar[index_current + 1] + 1) / 2 * ow) - 0.5;
        float ty_offset = ((points3d_polar[index_current + 2] + 1) / 2 * oh) - 0.5;

        float dx1 = points3d_polar[index_next_w + 1] - points3d_polar[index_current + 1];
        float dy1 = points3d_polar[index_next_w + 2] - points3d_polar[index_current + 2];
        float dx2 = points3d_polar[index_next_h + 1] - points3d_polar[index_current + 1];
        float dy2 = points3d_polar[index_next_h + 2] - points3d_polar[index_current + 2];
        float dx3 = points3d_polar[index_next_wh + 1] - points3d_polar[index_current + 1];
        float dy3 = points3d_polar[index_next_wh + 2] - points3d_polar[index_current + 2];

        float tx01 = dx1 / 2 * ow;
        float ty01 = dy1 / 2 * oh;
        float tx10 = dx2 / 2 * ow;
        float ty10 = dy2 / 2 * oh;
        float tx11 = dx3 / 2 * ow;
        float ty11 = dy3 / 2 * oh;

        float t00 = tx01 * -1.0/3 + tx10 *  2.0/3 + tx11 *  1.0/3;
        float t01 = ty01 * -1.0/3 + ty10 *  2.0/3 + ty11 *  1.0/3;
        float t10 = tx01 *  2.0/3 + tx10 * -1.0/3 + tx11 *  1.0/3;
        float t11 = ty01 *  2.0/3 + ty10 * -1.0/3 + ty11 *  1.0/3;

        float det = t00 * t11 - t01 * t10 + 1e-10;
        float it00 = t11 / det;
        float it01 = -t01 / det;
        float it10 = -t10 / det;
        float it11 = t00 / det;

        int this_depth = (int)(12800 / 128 * points3d_polar[index_current]);
        int delta_add = (int)(100 * depth_render[(int)(ty_offset * ow + tx_offset)]);
        int delta00 = this_depth - delta_add;
        int delta01 = (int)(12800 / 128 * points3d_polar[index_next_w]) - delta_add;
        int delta10 = (int)(12800 / 128 * points3d_polar[index_next_h]) - delta_add;
        int delta11 = (int)(12800 / 128 * points3d_polar[index_next_wh]) - delta_add;

        int mindelta = min(min(delta00, delta01), min(delta10, delta11));
        int maxdelta = max(max(delta00, delta01), max(delta10, delta11));

        int depth00 = this_depth;
        int depth01 = (int)(12800 / 128 * points3d_polar[index_next_w]);
        int depth10 = (int)(12800 / 128 * points3d_polar[index_next_h]);
        int depth11 = (int)(12800 / 128 * points3d_polar[index_next_wh]);
        int max_depth = max(max(depth00, depth10), max(depth01, depth11));
        int min_depth = min(min(depth00, depth10), min(depth01, depth11));
        int delta_depth = max_depth - min_depth;

        int txmin = floor(tx_offset + min(min(tx01, tx11), min(tx01, tx10)));
        int txmax = ceil(tx_offset + max(max(tx01, tx11), max(tx01, tx10)));
        int tymin = floor(ty_offset + min(min(ty01, ty11), min(ty01, ty10)));
        int tymax = ceil(ty_offset + max(max(ty01, ty11), max(ty01, ty10)));

        float tolerance = max(0.1 * this_depth, 10.0f);
        float tolerance2 = max(0.05 * max_depth, 10.0f);
        float flank = 0.01;

        if ((delta_depth < tolerance2) && (y > h / 8) && (y < 7 * h / 8)) {
            if ((mindelta > -tolerance) && (maxdelta < tolerance) && (this_depth < 10000)) {
                if (((txmax - txmin) * (tymax - tymin) < 1600) && (txmax - txmin < 40) && (tymax - tymin < 40)) {
                    for (int itx = txmin; itx < txmax; itx++) {
                        for (int ity = tymin; ity < tymax; ity++) {
                            if ((0 <= itx) && (itx < ow) && (0 <= ity) && (ity < oh)) {
                                float newx = (itx - tx_offset) * it00 + it10 * (ity - ty_offset);
                                float newy = (itx - tx_offset) * it01 + it11 * (ity - ty_offset);

                                if ((newx > -flank) && (newx < 1 + flank) && (newy > -flank) && (newy < 1 + flank)) {
                                    newx = clamp(newx, 0.0f, 1.0f);
                                    newy = clamp(newy, 0.0f, 1.0f);

                                    int r = img[(ih * w + iw)] / (256 * 256) * (1-newx) * (1-newy) +
                                            img[(ih * w + iw + 1)] / (256 * 256) * (1-newx) * newy +
                                            img[((ih + 1) * w + iw)] / (256 * 256) * newx * (1-newy) +
                                            img[((ih + 1) * w + iw + 1)] / (256 * 256) * newx * newy;

                                    int g = img[(ih * w + iw)] / 256 % 256 * (1-newx) * (1-newy) +
                                            img[(ih * w + iw + 1)] / 256 % 256 * (1-newx) * newy +
                                            img[((ih + 1) * w + iw)] / 256 % 256 * newx * (1-newy) +
                                            img[((ih + 1) * w + iw + 1)] / 256 % 256 * newx * newy;

                                    int b = img[(ih * w + iw)] % 256 * (1-newx) * (1-newy) +
                                            img[(ih * w + iw + 1)] % 256 * (1-newx) * newy +
                                            img[((ih + 1) * w + iw)] % 256 * newx * (1-newy) +
                                            img[((ih + 1) * w + iw + 1)] % 256 * newx * newy;

                                    r = min(r, 255);
                                    g = min(g, 255);
                                    b = min(b, 255);

                                    if ((ity * ow + itx > 0) && (ity * ow + itx < maxsize)) {
                                        render[(ity * ow + itx)] = r * 256 * 256 + g * 256 + b;
                                        selection[(ity * ow + itx)] = 1.0 / abs(det);
                                    }
                                }
                            }
                        }
                    }
                }
            }
        }
    }
}