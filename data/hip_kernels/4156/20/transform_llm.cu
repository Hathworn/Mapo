#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transform(float *points3d_after, float *points3d, float * transformation_matrix)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int w = gridDim.x * TILE_DIM;

    // Optimize loop to use shared memory to reduce global memory accesses.
    __shared__ float shared_transform[TILE_DIM][TILE_DIM][4];
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
        int iw = x;
        int ih = y + j;
        if (ih * w + iw < gridDim.y * TILE_DIM * TILE_DIM)
        {
            // Load data into shared memory
            for (int k = 0; k < 4; k++) {
                shared_transform[threadIdx.y + j][threadIdx.x][k] = points3d[(ih * w + iw) * 4 + k];
            }
            __syncthreads();

            // Compute transformation using shared memory
            for (int ic = 0; ic < 3; ic++) {
                float sum = 0.0f;
                for (int k = 0; k < 4; k++) {
                    sum += shared_transform[threadIdx.y + j][threadIdx.x][k] * transformation_matrix[4 * ic + k];
                }
                points3d_after[(ih * w + iw) * 3 + ic] = sum;
            }
            __syncthreads();
        }
    }
}