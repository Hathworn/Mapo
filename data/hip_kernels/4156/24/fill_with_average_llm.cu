#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_with_average(unsigned char *img, int *nz, int *average, int scale)
{
    // Calculate thread's global position
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    // Loop over tile dimension with stride of block size
    for (int j = y; j < y + TILE_DIM; j += BLOCK_ROWS)
    {
        int iw = x;
        int ih = j;

        // Only process pixels that meet condition
        if ((img[3 * (ih * width + iw)] + img[3 * (ih * width + iw) + 1] + img[3 * (ih * width + iw) + 2] == 0) && 
            (nz[ih / scale * width + iw / scale] > 0))
        {
            // Use shared memory to reduce bank access conflicts if needed (not used here)
            int index_scaled = (ih / scale) * width + iw / scale;
            int nz_value = nz[index_scaled];
            img[3 * (ih * width + iw)] = (unsigned char)(average[3 * index_scaled] / nz_value);
            img[3 * (ih * width + iw) + 1] = (unsigned char)(average[3 * index_scaled + 1] / nz_value);
            img[3 * (ih * width + iw) + 2] = (unsigned char)(average[3 * index_scaled + 2] / nz_value);
        }
    }
}