#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global function
__global__ void merge(unsigned char * img_all, unsigned char * img, float * selection, int n, int stride)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;
    float sum, weight;
    
    // Utilize shared memory for optimization
    __shared__ float shared_selection[TILE_DIM * BLOCK_ROWS];
    
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        sum = 0;
        
        // Aggregate selection values
        for (int idx = 0; idx < n; idx++) {
            shared_selection[threadIdx.y * TILE_DIM + threadIdx.x] = selection[idx * stride + ((y+j)*width + x)];
            sum += shared_selection[threadIdx.y * TILE_DIM + threadIdx.x];
        }

        // Normalize selection values
        for (int idx = 0; idx < n; idx++) {
            selection[idx * stride + ((y+j)*width + x)] = shared_selection[threadIdx.y * TILE_DIM + threadIdx.x] / (sum + 1e-5);
        }
        
        int pixel_position = 3 * ((y+j)*width + x);
        
        // Initialize pixel to zero
        img[pixel_position] = 0;
        img[pixel_position + 1] = 0;
        img[pixel_position + 2] = 0;
        
        // Compute weighted sum for image merging
        for (int idx = 0; idx < n; idx++) {
            // Use constant weight value for current implementation
            weight = 0.25;
            
            int img_position = idx * stride * 3 + pixel_position;
            
            // Accumulate weighted pixels
            img[pixel_position] += (unsigned char)(img_all[img_position] * weight);
            img[pixel_position + 1] += (unsigned char)(img_all[img_position + 1] * weight);
            img[pixel_position + 2] += (unsigned char)(img_all[img_position + 2] * weight);
        }
    }
}