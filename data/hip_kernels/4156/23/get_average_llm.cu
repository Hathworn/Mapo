#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void get_average(unsigned char * img, int * nz, int * average, int scale)
{
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
        int iw = x;
        int ih = y + j;

        int pixel_index = 3 * (ih * width + iw); // Compute base index only once
        int r = img[pixel_index];
        int g = img[pixel_index + 1];
        int b = img[pixel_index + 2];

        if (r + g + b > 0)
        {
            int scaled_index = ih / scale * width + iw / scale; // Compute scaled index only once
            atomicAdd(&(nz[scaled_index]), 1);
            atomicAdd(&(average[3 * scaled_index]), r);
            atomicAdd(&(average[3 * scaled_index + 1]), g);
            atomicAdd(&(average[3 * scaled_index + 2]), b);
        }
    }
}