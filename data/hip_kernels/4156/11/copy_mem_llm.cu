#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_mem(unsigned char *source, unsigned char *render)
{
    // Calculate x and y indices
    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    // Use shared memory for coalesced access
    __shared__ unsigned char sharedMem[TILE_DIM * (TILE_DIM + BLOCK_ROWS) * 3];
    
    // Load data into shared memory
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        for (int channel = 0; channel < 3; channel++) {
            int globalIndex = 3 * ((y + j) * width + x) + channel;
            int localIndex = 3 * (threadIdx.y * TILE_DIM + threadIdx.x + j) + channel;
            sharedMem[localIndex] = source[globalIndex];
        }
    }
    __syncthreads();

    // Copy from shared memory to global memory
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        for (int channel = 0; channel < 3; channel++) {
            int globalIndex = 3 * ((y + j) * width + x) + channel;
            int localIndex = 3 * (threadIdx.y * TILE_DIM + threadIdx.x + j) + channel;
            render[globalIndex] = sharedMem[localIndex];
        }
    }
}