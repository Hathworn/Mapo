#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Check if N is not zero to avoid division by zero
    if (N != 0) {
        // Perform atomic divide to safely handle concurrent writes
        atomicExch(accuracy, *accuracy / N);
    }
}