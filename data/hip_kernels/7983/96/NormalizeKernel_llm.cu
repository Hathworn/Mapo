#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;  // Index calculation
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < h && j < w)  // Boundary check
    {
        const int pos = i * s + j;  // Memory position calculation

        float scale = normalization_factor[pos];  // Read normalization factor

        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);  // Invert normalization factor

        image[pos] *= invScale;  // Normalize image
    }
}