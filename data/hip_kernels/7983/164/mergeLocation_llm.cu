#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Utilize registers to reduce memory accesses
        short2 loc = loc_[ptidx];
        float x_val = loc.x * scale;
        float y_val = loc.y * scale;

        // Write results back to global memory
        x[ptidx] = x_val;
        y[ptidx] = y_val;
    }
}