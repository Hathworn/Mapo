#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Compute target pixel position
    float cx = u[flow_row_offset + j] * time_scale + j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + i + 1.0f;

    // Decompose to integer and fractional parts
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    // Target pixel integer coordinates
    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];
    float weight;

    // Compute weights and update the corresponding pixels
    float weights[4] = {
        dx * dy,
        (1.0f - dx) * dy,
        (1.0f - dx) * (1.0f - dy),
        dx * (1.0f - dy)
    };

    int offsets[4][2] = {
        {0,  0},
        {-1, 0},
        {-1, -1},
        {0, -1}
    };

    #pragma unroll
    for (int k = 0; k < 4; ++k) {
        int cur_tx = tx + offsets[k][0];
        int cur_ty = ty + offsets[k][1];
        if ((cur_tx >= 0) && (cur_tx < w) && (cur_ty >= 0) && (cur_ty < h)) {
            int idx = cur_ty * image_stride + cur_tx;
            _atomicAdd(dst + idx, value * weights[k]);
            _atomicAdd(normalization_factor + idx, weights[k]);
        }
    }
}