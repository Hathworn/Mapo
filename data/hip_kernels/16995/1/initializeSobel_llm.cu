#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define GAUSSIAN_KERNEL_SIZE 3
#define SOBEL_KERNEL_SIZE 5
#define TILE_WIDTH 32
#define SMEM_SIZE 128

__global__ void initializeSobel(float *d_sobelKernelX, float *d_sobelKernelY) {
    int ix = threadIdx.x;
    int iy = threadIdx.y;
    
    // Calculate workload directly based on limits of the kernel size
    if (ix < SOBEL_KERNEL_SIZE && iy < SOBEL_KERNEL_SIZE) {
        int index = iy * SOBEL_KERNEL_SIZE + ix;
        
        // Use constant memory for reusable values
        const float weight = (float)(SOBEL_KERNEL_SIZE / 2);
        float sx = (float)(ix - weight);
        float sy = (float)(iy - weight);
        float norm = sx * sx + sy * sy;

        // Branch optimization: handle division outside conditional
        float sx_norm_weight = sx / norm * weight;
        float sy_norm_weight = sy / norm * weight;
        
        d_sobelKernelX[index] = (norm == 0.0f) ? 0.0f : sx_norm_weight;
        d_sobelKernelY[index] = (norm == 0.0f) ? 0.0f : sy_norm_weight;
    }
}