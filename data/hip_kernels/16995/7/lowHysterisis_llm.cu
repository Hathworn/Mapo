#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define GAUSSIAN_KERNEL_SIZE 3
#define SOBEL_KERNEL_SIZE 5
#define TILE_WIDTH 32
#define SMEM_SIZE 128

__global__ void lowHysterisis(int width, int height, float *d_nonMax, float* d_highThreshHyst, float lowThreshold, float *d_lowThreshHyst) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if ((ix > 0) && (ix < (width - 1)) && (iy > 0) && iy < (height - 1)) {
        int tid = iy * width + ix;

        d_lowThreshHyst[tid] = d_highThreshHyst[tid];

        if (d_highThreshHyst[tid] == 1) {
            // Precompute neighbor indices
            int neighbors[8] = {
                tid + 1,       // eastN
                tid - 1,       // westN
                tid - width,   // northN
                tid + width,   // southN
                tid + width + 1,  // southEastN
                tid - width + 1,  // northEastN
                tid + width - 1,  // southWestN
                tid - width - 1   // northWestN
            };
            
            // Loop through neighbors
            for (int i = 0; i < 8; ++i) {
                if (d_nonMax[neighbors[i]] > lowThreshold) {
                    d_lowThreshHyst[neighbors[i]] = 1.0f;
                }
            }
        }
    }
}