#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

# define MAX(a, b) ((a) > (b) ? (a) : (b))

# define GAUSSIAN_KERNEL_SIZE 3
# define SOBEL_KERNEL_SIZE 5
# define TILE_WIDTH 32
# define SMEM_SIZE 128

__global__ void highHysterisis(int width, int height, float* d_nonMax, float highThreshold, float *d_highThreshHyst) {
    // Calculate unique thread index in the global memory
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Calculate the linear index in the array
    int tid = iy * width + ix;

    // Check bounds to avoid accessing invalid memory
    if (ix < width && iy < height) {
        // Perform thresholding
        d_highThreshHyst[tid] = d_nonMax[tid] > highThreshold ? 1.0f : 0.0f;
    }
}