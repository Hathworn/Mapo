#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX(a, b) ((a) > (b) ? (a) : (b))

#define GAUSSIAN_KERNEL_SIZE 3
#define SOBEL_KERNEL_SIZE 5
#define TILE_WIDTH 32
#define SMEM_SIZE 128

__global__ void nonMaxSuppressionDevice(int width, int height, float *d_gradientX, float *d_gradientY, float* d_gradientMag, float* d_nonMax) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix < width && iy < height) {
        int tid = iy * width + ix;

        // Read gradient magnitudes and gradients
        float d_gradientMag_tid = d_gradientMag[tid];
        float d_gradientXT = d_gradientX[tid];
        float d_gradientYT = d_gradientY[tid];

        float tanYX, magB, magA;

        // Handle boundary conditions: top/bottom edges and left/right edges
        if (ix == 0 || ix == width - 1 || iy == 0 || iy == height - 1) {
            d_nonMax[tid] = 0;
        } else {
            if (d_gradientMag_tid == 0) {
                d_nonMax[tid] = 0;
            } else {
                int offset1, offset2;
                float tanYX, magA, magB;
                
                // Calculate direction and potential offsets
                if (d_gradientXT >= 0) {
                    if (d_gradientYT >= 0) {
                        if (d_gradientXT >= d_gradientYT) { // East of South-East
                            tanYX = d_gradientYT / d_gradientXT;
                            offset1 = 1;
                            offset2 = width + 1;
                        } else { // South of South-East
                            tanYX = d_gradientXT / d_gradientYT;
                            offset1 = width;
                            offset2 = width + 1;
                        }
                    } else {
                        if (d_gradientXT >= -d_gradientYT) { // East of North-East
                            tanYX = -d_gradientYT / d_gradientXT;
                            offset1 = 1;
                            offset2 = -width + 1;
                        } else { // North of North-East
                            tanYX = d_gradientXT / -d_gradientYT;
                            offset1 = width;
                            offset2 = width - 1;
                        }
                    }
                } else {
                    if (d_gradientYT >= 0) {
                        if (d_gradientYT >= -d_gradientXT) { // South of South-West
                            tanYX = -d_gradientXT / d_gradientYT;
                            offset1 = width;
                            offset2 = width - 1;
                        } else { // West of South-West
                            tanYX = d_gradientYT / -d_gradientXT;
                            offset1 = -1;
                            offset2 = width - 1;
                        }
                    } else {
                        if (d_gradientYT >= d_gradientXT) { // West of North-West
                            tanYX = d_gradientYT / d_gradientXT;
                            offset1 = -1;
                            offset2 = -width - 1;
                        } else { // North of North-West
                            tanYX = d_gradientXT / d_gradientYT;
                            offset1 = width;
                            offset2 = width + 1;
                        }
                    }
                }

                // Compute magnitude interpolations
                magA = (1 - tanYX) * d_gradientMag[tid + offset1] + tanYX * d_gradientMag[tid + offset2];
                magB = (1 - tanYX) * d_gradientMag[tid - offset1] + tanYX * d_gradientMag[tid - offset2];

                // Perform non-maximum suppression
                d_nonMax[tid] = (d_gradientMag_tid >= magA && d_gradientMag_tid >= magB) ? d_gradientMag_tid : 0;
            }
        }
    }
}