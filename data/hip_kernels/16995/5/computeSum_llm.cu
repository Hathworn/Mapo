#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX(a, b) ((a) > (b) ? (a) : (b))

#define GAUSSIAN_KERNEL_SIZE 3
#define SOBEL_KERNEL_SIZE 5
#define TILE_WIDTH 32
#define SMEM_SIZE 128

__global__ void computeSum(float *d_filteredImage, float *d_imageSumGrid, unsigned int n)
{
    __shared__ float smem[SMEM_SIZE];
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    float localSum = 0;

    // Unroll the loop for better performance and memory coalescing
    if (idx + 3 * blockDim.x < n) {
        #pragma unroll
        for (int i = 0; i < 4; i++) {
            localSum += d_filteredImage[idx + i * blockDim.x];
        }
    }

    smem[tid] = localSum;
    __syncthreads();

    // Replace multiple syncthreads with a single loop
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            smem[tid] += smem[tid + stride];
        }
        __syncthreads();
    }

    if (tid < 32) {
        volatile float *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    if (tid == 0) {
        d_imageSumGrid[blockIdx.x] = smem[0];
    }
}