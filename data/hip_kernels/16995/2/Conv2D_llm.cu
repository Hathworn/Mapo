#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX(a, b) ((a) > (b) ? (a) : (b))

#define GAUSSIAN_KERNEL_SIZE 3
#define SOBEL_KERNEL_SIZE 5
#define TILE_WIDTH 32
#define SMEM_SIZE 128

__global__ void Conv2D(float *d_image, float *kernel, float *d_result, int width, int height, int kernelSize) {
    const int sharedMemWidth = TILE_WIDTH + MAX(SOBEL_KERNEL_SIZE, GAUSSIAN_KERNEL_SIZE) - 1;
    __shared__ float sharedMem[sharedMemWidth][sharedMemWidth];

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    // Calculate global and local indices
    int x = blockIdx.x * TILE_WIDTH + tx;
    int y = blockIdx.y * TILE_WIDTH + ty;
    
    // Load data into shared memory
    for (int by = 0; by <= 1; ++by) {
        for (int bx = 0; bx <= 1; ++bx) {
            int destX = tx + bx * TILE_WIDTH;
            int destY = ty + by * TILE_WIDTH;
            int srcX = x + bx * TILE_WIDTH - kernelSize / 2;
            int srcY = y + by * TILE_WIDTH - kernelSize / 2;
            
            // Boundary check for loading from global memory
            if (destY < sharedMemWidth && srcY >= 0 && srcY < height && srcX >= 0 && srcX < width) {
                sharedMem[destY][destX] = d_image[srcY * width + srcX];
            } else if (destY < sharedMemWidth) {
                sharedMem[destY][destX] = 0;
            }
        }
    }
   
    __syncthreads();

    // Perform convolution
    float accum = 0;
    for (int j = 0; j < kernelSize; j++) {
        for (int i = 0; i < kernelSize; i++) {
            accum += sharedMem[ty + j][tx + i] * kernel[j * kernelSize + i];
        }
    }

    // Write result to output
    if (x < width && y < height) {
        d_result[y * width + x] = fminf(fmaxf(accum, 0.0f), 1.0f);
    }

    __syncthreads();
}