#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX(a, b) ((a) > (b) ? (a) : (b))

#define GAUSSIAN_KERNEL_SIZE 3
#define SOBEL_KERNEL_SIZE 5
#define TILE_WIDTH 32
#define SMEM_SIZE 128

__global__ void magnitudeImage(float *d_gradientX, float *d_gradientY, float *d_gradientMag, int width, int height) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix < width && iy < height) {
        int idx = iy * width + ix;
        
        // Use hypotf instead of powf and sqrtf for better performance
        d_gradientMag[idx] = hypotf(d_gradientX[idx], d_gradientY[idx]);
    }
}