#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Use blockDim and blockIdx for cleaner indexing
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;
    
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}