#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to find 3D minimum and maximum values
__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int pitch, int height)
{
    // Shared memory for data caching
    __shared__ float data1[3*(MINMAX_W + 2)];
    __shared__ float data2[3*(MINMAX_W + 2)];
    __shared__ float data3[3*(MINMAX_W + 2)];
    __shared__ float ymin1[(MINMAX_W + 2)];
    __shared__ float ymin2[(MINMAX_W + 2)];
    __shared__ float ymin3[(MINMAX_W + 2)];
    __shared__ float ymax1[(MINMAX_W + 2)];
    __shared__ float ymax2[(MINMAX_W + 2)];
    __shared__ float ymax3[(MINMAX_W + 2)];

    // Current tile and apron limits, relative to row start
    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_W;
    const int xEnd = xStart + MINMAX_W - 1;
    const int xReadPos = xStart + tx - WARP_SIZE;
    const int xWritePos = xStart + tx;
    const int xEndClamped = min(xEnd, width - 1);
    int memWid = MINMAX_W + 2;

    int memPos0 = (tx - WARP_SIZE + 1);
    int memPos1 = (tx - WARP_SIZE + 1);
    int yq = 0;
    unsigned int output = 0;

    for (int y = 0; y < 32 + 2; y++) {
        output >>= 1;
        int memPos = yq * memWid + (tx - WARP_SIZE + 1);
        int yp = 32 * blockIdx.y + y - 1;
        yp = max(yp, 0);
        yp = min(yp, height - 1);
        int readStart = yp * pitch;

        // Use conditional operator for cleaner code
        if (tx >= (WARP_SIZE - 1)) {
            data1[memPos] = (xReadPos < 0 || xReadPos >= width) ? 0 : d_Data1[readStart + xReadPos];
            data2[memPos] = (xReadPos < 0 || xReadPos >= width) ? 0 : d_Data2[readStart + xReadPos];
            data3[memPos] = (xReadPos < 0 || xReadPos >= width) ? 0 : d_Data3[readStart + xReadPos];
        }
        __syncthreads();

        int memPos2 = yq * memWid + tx;
        if (y > 1) {
            if (tx < memWid) {
                // Reduce redundant function calls for better performance
                float min1 = fminf(data1[memPos0], data1[memPos1]);
                min1 = fminf(min1, data1[memPos2]);
                float min2 = fminf(data2[memPos0], data2[memPos1]);
                min2 = fminf(min2, data2[memPos2]);
                float min3 = fminf(data3[memPos0], data3[memPos1]);
                min3 = fminf(min3, data3[memPos2]);
                float max1 = fmaxf(data1[memPos0], data1[memPos1]);
                max1 = fmaxf(max1, data1[memPos2]);
                float max2 = fmaxf(data2[memPos0], data2[memPos1]);
                max2 = fmaxf(max2, data2[memPos2]);
                float max3 = fmaxf(data3[memPos0], data3[memPos1]);
                max3 = fmaxf(max3, data3[memPos2]);
                
                // Consolidate calculations for ymin and ymax
                ymin1[tx] = min1;
                ymin2[tx] = fminf(fminf(min1, min2), min3);
                ymin3[tx] = min3;
                ymax1[tx] = max1;
                ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
                ymax3[tx] = max3;
            }
        }
        __syncthreads();

        if (y > 1) {
            if (tx < MINMAX_W) {
                if (xWritePos <= xEndClamped) {
                    // Simplify minv and maxv calculations
                    float minv = fminf(ymin2[tx], ymin2[tx+2]);
                    minv = fminf(minv, ymin1[tx+1]);
                    minv = fminf(minv, ymin3[tx+1]);
                    minv = fminf(minv, data2[memPos0+1]);
                    minv = fminf(minv, data2[memPos2+1]);
                    minv = fminf(minv, d_Threshold[1]);

                    float maxv = fmaxf(ymax2[tx], ymax2[tx+2]);
                    maxv = fmaxf(maxv, ymax1[tx+1]);
                    maxv = fmaxf(maxv, ymax3[tx+1]);
                    maxv = fmaxf(maxv, data2[memPos0+1]);
                    maxv = fmaxf(maxv, data2[memPos2+1]);
                    maxv = fmaxf(maxv, d_Threshold[0]);

                    // Update output based on threshold
                    if (data2[memPos1+1] < minv || data2[memPos1+1] > maxv)
                        output |= 0x80000000;
                }
            }
        }
        __syncthreads();

        memPos0 = memPos1;
        memPos1 = memPos2;
        yq = (yq < 2 ? yq + 1 : 0);
    }

    // Write the output to the global memory
    if (tx < MINMAX_W && xWritePos < width) {
        int writeStart = blockIdx.y * pitch + xWritePos;
        d_Result[writeStart] = output;
    }
}