#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Removed unnecessary __syncthreads() for better performance
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim.x for dynamic block size
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Use blockDim.y for dynamic block size
    int p = y * pitch + x;
    if (x < width && y < height)
        d_Result[p] = d_Data1[p] - d_Data2[p];
}