#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    int p = y * width + x;

    // Initializing shared memory
    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {  // Handle out-of-bound access
        float val = d_Data[p - x];
        minvals[tx] = val;
        maxvals[tx] = val;
    }

    // Loop over rows to find local min/max
    for (int ty = 1; ty < 16; ty++) {
        p += width;
        if (x < width) {
            float val = d_Data[p];
            minvals[tx] = min(minvals[tx], val);
            maxvals[tx] = max(maxvals[tx], val);
        }
    }
    __syncthreads();

    // Reduction to find block min/max using binary reduction
    for (int d = 1; d < blockDim.x; d *= 2) {
        if (tx % (2 * d) == 0) {
            minvals[tx] = min(minvals[tx], minvals[tx + d]);
            maxvals[tx] = max(maxvals[tx], maxvals[tx + d]);
        }
        __syncthreads();
    }

    // Write block min/max to global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr + 0] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}