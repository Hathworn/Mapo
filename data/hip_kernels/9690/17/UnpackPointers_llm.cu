#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    // Cache threadIdx.x value
    const int tx = threadIdx.x;
    // Loop unrolling to reduce branch overhead
    int numPts = 0;
    const int stride = 32;
    const int widthStride = w / 16;

    for (int y = 0; y < h / stride; y++) {
        for (int x = 0; x < w; x += 16) {
            unsigned int val = minmax[y * w + x + tx];
            if (val) {
                #pragma unroll
                for (int k = 0; k < stride; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        // Efficient calculation without redundant operations
                        ptrs[16 * numPts + tx] = (y * stride + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}