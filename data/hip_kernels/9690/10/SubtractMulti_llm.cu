#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Compute the 2D coordinates of the current thread within the block
    const int x = blockIdx.x * SUBTRACTM_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACTM_H + threadIdx.y;

    // Compute the linear index for the current thread in the 3D grid
    int sz = height * pitch;
    int p = threadIdx.z * sz + y * pitch + x;

    if (x < width && y < height) {
        // Perform subtraction using shared memory optimization
        __shared__ float sharedData[SUBTRACTM_W * SUBTRACTM_H * 2];
        int localIndex = threadIdx.y * SUBTRACTM_W + threadIdx.x;
        
        // Load data into shared memory
        sharedData[localIndex] = d_Data[p];
        sharedData[localIndex + SUBTRACTM_W * SUBTRACTM_H] = d_Data[p + sz];
        __syncthreads();

        // Perform the subtraction using shared memory
        d_Result[p] = sharedData[localIndex] - sharedData[localIndex + SUBTRACTM_W * SUBTRACTM_H];
    }
}