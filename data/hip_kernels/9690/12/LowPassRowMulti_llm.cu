#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LowPassRowMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    __shared__ float data[CONVROW_W + 2*RADIUS];

    const int tx = threadIdx.x;
    const int block = blockIdx.x/(NUM_SCALES+3);
    const int scale = blockIdx.x - (NUM_SCALES+3)*block;
    const int xout = block*CONVROW_W + tx;
    const int loadPos = xout - RADIUS;
    const int yptr = blockIdx.y*pitch;
    const int writePos = yptr + height*pitch*scale + xout;
    float *kernel = d_Kernel + scale*16;

    // Load data conditionally, reducing load/store divergence
    data[tx] = d_Data[yptr + max(0, min(width-1, loadPos))];
    
    __syncthreads();

    // Directly compute and streamline conditional
    if (xout < width && tx < CONVROW_W)
    {
        float sum = 0.0f;
        #pragma unroll // Unroll the loop to improve performance
        for (int i = 0; i <= 4; i++)
        {
            sum += (data[tx + i] + data[tx + (8-i)]) * kernel[i];
        }
        d_Result[writePos] = sum;
    }

    __syncthreads();
}