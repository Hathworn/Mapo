#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int height) {
    // Data cache
    extern __shared__ float shared_data[];
    float *data1 = shared_data;
    float *data2 = &shared_data[3 * (MINMAX_SIZE + 2)];
    float *data3 = &shared_data[6 * (MINMAX_SIZE + 2)];
    float *ymin1 = &shared_data[9 * (MINMAX_SIZE + 2)];
    float *ymin2 = &shared_data[10 * (MINMAX_SIZE + 2)];
    float *ymin3 = &shared_data[11 * (MINMAX_SIZE + 2)];
    float *ymax1 = &shared_data[12 * (MINMAX_SIZE + 2)];
    float *ymax2 = &shared_data[13 * (MINMAX_SIZE + 2)];
    float *ymax3 = &shared_data[14 * (MINMAX_SIZE + 2)];

    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_SIZE;
    const int xEnd = xStart + MINMAX_SIZE - 1;
    const int xReadPos = xStart + tx - WARP_SIZE;
    const int xWritePos = xStart + tx;
    const int xEndClamped = min(xEnd, width - 1);
    int memWid = MINMAX_SIZE + 2;
    int memPos0 = (tx - WARP_SIZE + 1);
    int memPos1 = (tx - WARP_SIZE + 1);
    int yq = 0;
    unsigned int output = 0;

    for (int y = 0; y < 34; y++) {
        output >>= 1;
        int memPos = yq * memWid + (tx - WARP_SIZE + 1);
        int yp = 32 * blockIdx.y + y - 1;
        yp = max(yp, 0);
        yp = min(yp, height - 1);
        int readStart = yp * width;

        // Load data into shared memory
        if (tx >= (WARP_SIZE - 1)) {
            if (xReadPos < 0 || xReadPos >= width) {
                data1[memPos] = 0;
                data2[memPos] = 0;
                data3[memPos] = 0;
            } else {
                data1[memPos] = d_Data1[readStart + xReadPos];
                data2[memPos] = d_Data2[readStart + xReadPos];
                data3[memPos] = d_Data3[readStart + xReadPos];
            }
        }
        __syncthreads();

        int memPos2 = yq * memWid + tx;
        if (y > 1 && tx < memWid) {
            // Compute min/max values
            float min1 = fminf(fminf(data1[memPos0], data1[memPos1]), data1[memPos2]);
            float min2 = fminf(fminf(data2[memPos0], data2[memPos1]), data2[memPos2]);
            float min3 = fminf(fminf(data3[memPos0], data3[memPos1]), data3[memPos2]);
            float max1 = fmaxf(fmaxf(data1[memPos0], data1[memPos1]), data1[memPos2]);
            float max2 = fmaxf(fmaxf(data2[memPos0], data2[memPos1]), data2[memPos2]);
            float max3 = fmaxf(fmaxf(data3[memPos0], data3[memPos1]), data3[memPos2]);

            ymin1[tx] = min1;
            ymin2[tx] = fminf(fminf(min1, min2), min3);
            ymin3[tx] = min3;
            ymax1[tx] = max1;
            ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
            ymax3[tx] = max3;
        }
        __syncthreads();

        if (y > 1 && tx < MINMAX_SIZE) {
            if (xWritePos <= xEndClamped) {
                // Compute final min/max and update output
                float minv = fminf(fminf(fminf(fminf(fminf(ymin2[tx], ymin2[tx + 2]), ymin1[tx + 1]), ymin3[tx + 1]), data2[memPos0 + 1]), data2[memPos2 + 1]);
                minv = fminf(minv, d_ConstantA[1]);
                float maxv = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(ymax2[tx], ymax2[tx + 2]), ymax1[tx + 1]), ymax3[tx + 1]), data2[memPos0 + 1]), data2[memPos2 + 1]);
                maxv = fmaxf(maxv, d_ConstantA[0]);
                if (data2[memPos1 + 1] < minv || data2[memPos1 + 1] > maxv)
                    output |= 0x80000000;
            }
        }
        __syncthreads();

        memPos0 = memPos1;
        memPos1 = memPos2;
        yq = (yq < 2 ? yq + 1 : 0);
    }

    if (tx < MINMAX_SIZE && xWritePos < width) {
        int writeStart = blockIdx.y * width + xWritePos;
        d_Result[writeStart] = output;
    }
}