#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float *A, const float *B, float *C, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory for faster access
    __shared__ float sA[256];
    __shared__ float sB[256];

    // Load data into shared memory
    if (i < n) {
        sA[threadIdx.x] = A[i];
        sB[threadIdx.x] = B[i];
    }
    __syncthreads();

    // Perform computation using shared memory
    if (i < n)
    {
        C[i] = sA[threadIdx.x] + sB[threadIdx.x];
    }
}