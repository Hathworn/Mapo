#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float getPixelValue(float* img, int x, int y, int width, int height) {
    if (x >= 0 && x < width && y >= 0 && y < height) {
        return img[y * width + x];
    }
    return 0.0f;
}

__global__ void imgBlur(float* imgIn, float* imgOut, int imageWidth, int imageHeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check within bounds
    if(idx < imageWidth && idy < imageHeight)
    {
        float sum = 0.0f;
        // Optimize access by pre-calculating positions
        sum += getPixelValue(imgIn, idx, idy, imageWidth, imageHeight);
        sum += getPixelValue(imgIn, idx-1, idy-1, imageWidth, imageHeight);
        sum += getPixelValue(imgIn, idx-1, idy, imageWidth, imageHeight);
        sum += getPixelValue(imgIn, idx+1, idy, imageWidth, imageHeight);
        sum += getPixelValue(imgIn, idx+1, idy+1, imageWidth, imageHeight);
        sum += getPixelValue(imgIn, idx+1, idy-1, imageWidth, imageHeight);
        sum += getPixelValue(imgIn, idx, idy-1, imageWidth, imageHeight);
        sum += getPixelValue(imgIn, idx, idy+1, imageWidth, imageHeight);
        sum += getPixelValue(imgIn, idx-1, idy+1, imageWidth, imageHeight);

        imgOut[idy * imageWidth + idx] = sum / (float)(BLUR_SIZE * BLUR_SIZE);
    }
}