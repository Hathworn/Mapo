#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixAdd(float *A, float *B, float *C, int n)
{
    // Compute global row and column indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n)
    {
        // Flatten row and column into a single index
        int i = row * n + col;
        // Perform addition
        C[i] = A[i] + B[i];
    }
}