#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMul(int *A, int *B, int *C, int n) {
    // Calculate the global row and column indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if row and column are within matrix bounds
    if (row < n && col < n) {
        // Use register variable for temporary value
        int Cvalue = 0;
        // Loop to perform the dot product
        for (int i = 0; i < n; i++) {
            Cvalue += A[row * n + i] * B[i * n + col];
        }
        // Write the result back to the global memory
        C[row * n + col] = Cvalue;
    }
}