#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16  // Ensure TILE_WIDTH is defined

__global__ void matrix_multiply(float *a, float *b, float *c, int num, size_t width)
{
    // Thread and block index shorthand
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;

    // Allocate tiles in shared memory
    __shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
    __shared__ float s_b[TILE_WIDTH][TILE_WIDTH];

    // Row and column index calculations
    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    float result = 0;

    // Loop over tiles
    for (int i = 0; i < (width + TILE_WIDTH - 1) / TILE_WIDTH; ++i) // Optimized loop end condition
    {
        // Load tiles into shared memory with boundary checks
        if (row < width && i * TILE_WIDTH + tx < width)
        {
            s_a[ty][tx] = a[row * width + i * TILE_WIDTH + tx];
        }
        else
        {
            s_a[ty][tx] = 0.0f; // Ensure initialization to 0.0f
        }

        if (col < width && i * TILE_WIDTH + ty < width)
        {
            s_b[ty][tx] = b[(i * TILE_WIDTH + ty) * width + col];
        }
        else
        {
            s_b[ty][tx] = 0.0f; // Ensure initialization to 0.0f
        }

        __syncthreads(); // Sync threads after loading tiles

        // Dot product computation
        for (int k = 0; k < TILE_WIDTH; ++k)
        {
            result += s_a[ty][k] * s_b[k][tx];
        }

        __syncthreads(); // Sync threads before loading next tiles
    }

    // Write the result to global memory
    if (row < width && col < width)
    {
        c[row * num + col] = result; // Correct index calculation
    }
}