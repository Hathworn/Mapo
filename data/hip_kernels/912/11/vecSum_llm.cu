#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecSum(double* devIn, int pow_step, int n)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure threads do not access out-of-bounds memory
    int index = pow_step * i;
    if(index + (pow_step / 2) < n)
    {
        // Combine elements and store the result in the first element
        devIn[index] += devIn[index + (pow_step / 2)];
    }
}