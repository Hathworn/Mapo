#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(float *x, float *y, float *z, float *deltaX, float *deltaY, float *deltaZ) {
    // More efficient indexing using built-in thread and block index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  
    if (tid < N) {
        x[tid] += deltaX[tid];
        y[tid] += deltaY[tid];
        z[tid] += deltaZ[tid];
    }
}