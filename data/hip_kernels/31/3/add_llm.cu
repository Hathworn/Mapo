#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add( int *a, int *b, int *c ) {
    // Use blockIdx.x * blockDim.x + threadIdx.x for unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use total threads to stride through the array
    int totalThreads = gridDim.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += totalThreads;
    }
}