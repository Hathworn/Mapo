#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread id
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}