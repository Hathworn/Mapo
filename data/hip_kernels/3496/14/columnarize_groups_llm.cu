#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void columnarize_groups(int8_t* columnar_buffer, const int8_t* rowwise_buffer, const size_t row_count, const size_t col_count, const size_t* col_widths, const size_t row_size) {
    // Calculate 1D flattened thread index
    const auto thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread index is within bounds
    if (thread_index >= row_count) {
        return;
    }
    
    // Calculate pointers for reading and writing
    auto read_ptr = rowwise_buffer + thread_index * row_size;
    
    // Loop through each column
    for (size_t i = 0; i < col_count; ++i) {
        auto col_base = columnar_buffer + i * row_count * col_widths[i];
        
        // Use shared memory to reduce redundant calculations (if needed to optimize further)
        switch (col_widths[i]) {
            case 8: {
                // Write in chunks of int64
                reinterpret_cast<int64_t*>(col_base)[thread_index] = *reinterpret_cast<const int64_t*>(read_ptr);
                break;
            }
            case 4: {
                // Write in chunks of int32
                reinterpret_cast<int32_t*>(col_base)[thread_index] = *reinterpret_cast<const int32_t*>(read_ptr);
                break;
            }
            // Additional case optimization can be considered if col_widths has other frequent values
            default: break;
        }

        // Move read pointer forward by column width
        read_ptr += col_widths[i];
    }
}