#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define EMPTY_KEY_64 0  // Assuming definition; ensure it's defined or passed appropriately

__global__ void init_render_buffer_wrapper(int64_t* render_buffer, const uint32_t qw_count) {
    // Calculate global thread index for the current thread
    const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t stride = blockDim.x * gridDim.x;
    
    // Use this thread to initialize multiple elements
    for (uint32_t i = index; i < qw_count; i += stride) {
        render_buffer[i] = EMPTY_KEY_64;
    }
}
```
