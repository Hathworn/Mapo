#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_group(int8_t* groups, const size_t group_count, const size_t col_count, const size_t* col_widths, const size_t* init_vals) {
    // Calculate the global thread index
    const size_t thread_index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Early return if the thread index is out of bounds
    if (thread_index >= group_count) return;

    // Use a temporary pointer for efficiency
    int8_t* group_ptr = groups + thread_index;

    for (size_t i = 0; i < col_count; ++i) {
        // Check the column width and initialize accordingly
        if (col_widths[i] == 4) {
            *reinterpret_cast<uint32_t*>(group_ptr) = *reinterpret_cast<const uint32_t*>(&init_vals[i]);
        } else if (col_widths[i] == 8) {
            *reinterpret_cast<size_t*>(group_ptr) = init_vals[i];
        }
        // Move the group pointer by the current column width times the group count
        group_ptr += col_widths[i] * group_count;
    }
}