#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_valid_pos_flag(int32_t* pos_buff, const int32_t* count_buff, const int32_t entry_count) {
    // Use shared memory to reduce global memory accesses
    __shared__ int32_t shared_buff[1024];
    
    const int32_t start = threadIdx.x + blockDim.x * blockIdx.x;
    const int32_t step = blockDim.x * gridDim.x;
    
    for (int32_t i = start; i < entry_count; i += step) {
        // Load data to shared memory from global memory
        shared_buff[threadIdx.x] = count_buff[i];
        __syncthreads();
        
        // Check condition and update pos_buff
        if (shared_buff[threadIdx.x]) {
            pos_buff[i] = VALID_POS_FLAG;
        }
        __syncthreads();
    }
}