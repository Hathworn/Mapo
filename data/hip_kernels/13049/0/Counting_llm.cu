#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Counting(int* HalfData, int HalfDataSize, int N)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop unrolling for better performance
    int stride = blockDim.x * gridDim.x;
    for (int idx = i; idx < HalfDataSize; idx += stride) {
        HalfData[idx] *= N;
    }
}