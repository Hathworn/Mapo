#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello(char *a, int *b)
{
    // Use blockDim to parallelize the addition operation
    int idx = threadIdx.x; // Each thread processes one element
    if(idx < 7) // Ensure no out-of-bounds access
    {
        a[idx] += b[idx];
    }
}