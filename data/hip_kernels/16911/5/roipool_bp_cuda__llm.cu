#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void roipool_bp_cuda_(int nProposal, int C, float *d_feats, int *proposals_offset, int *output_maxidx, float *d_output_feats) {
    // Use shared memory for faster access
    extern __shared__ int shared_output_maxidx[];

    int plane = threadIdx.x;

    for (int pp_id = blockIdx.x; pp_id < nProposal; pp_id += gridDim.x) {
        // Load data into shared memory
        if (plane < C) {
            shared_output_maxidx[plane] = output_maxidx[pp_id * C + plane];
        }
        __syncthreads();

        if (plane < C) {
            int argmax_idx = shared_output_maxidx[plane];
            atomicAdd(&d_feats[argmax_idx * C + plane], d_output_feats[pp_id * C + plane]);
        }
        __syncthreads();
    }
}