#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* ================================== ballquery_batch_p ================================== */

__global__ void ballquery_batch_p_cuda_(int n, int meanActive, float radius, const float *xyz, const int *batch_idxs, const int *batch_offsets, int *idx, int *start_len, int *cumsum) {
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= n) return;

    start_len += (pt_idx * 2);
    int idx_temp[1000];

    float radius2 = radius * radius;
    float o_x = xyz[pt_idx * 3 + 0];
    float o_y = xyz[pt_idx * 3 + 1];
    float o_z = xyz[pt_idx * 3 + 2];

    int batch_idx = batch_idxs[pt_idx];
    int start = batch_offsets[batch_idx];
    int end = batch_offsets[batch_idx + 1];

    int cnt = 0;

    // Optimize: Use shared memory to cache xyz data for better memory access pattern
    extern __shared__ float shared_xyz[];
    for(int k = start + threadIdx.x; k < end; k += blockDim.x) {
        shared_xyz[threadIdx.x * 3 + 0] = xyz[k * 3 + 0];
        shared_xyz[threadIdx.x * 3 + 1] = xyz[k * 3 + 1];
        shared_xyz[threadIdx.x * 3 + 2] = xyz[k * 3 + 2];
        __syncthreads();

        for(int j = 0; j < blockDim.x && (start + j) < end; ++j) {
            float x = shared_xyz[j * 3 + 0];
            float y = shared_xyz[j * 3 + 1];
            float z = shared_xyz[j * 3 + 2];
            float d2 = (o_x - x) * (o_x - x) + (o_y - y) * (o_y - y) + (o_z - z) * (o_z - z);
            if(d2 < radius2) {
                if(cnt < 1000) {
                    idx_temp[cnt] = start + j;
                }
                else{
                    break;
                }
                ++cnt;
            }
        }
        __syncthreads();
    }

    start_len[0] = atomicAdd(cumsum, cnt);
    start_len[1] = cnt;

    int thre = n * meanActive;
    if(start_len[0] >= thre) return;

    idx += start_len[0];
    if(start_len[0] + cnt >= thre) cnt = thre - start_len[0];

    for(int k = 0; k < cnt; k++) {
        idx[k] = idx_temp[k];
    }
}