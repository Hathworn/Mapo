#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void get_iou_cuda_(int nInstance, int nProposal, int *proposals_idx, int *proposals_offset, long *instance_labels, int *instance_pointnum, float *proposals_iou) {
    int proposal_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (proposal_id < nProposal) {  // Ensure valid proposal_id
        int start = proposals_offset[proposal_id];
        int end = proposals_offset[proposal_id + 1];
        int proposal_total = end - start;

        extern __shared__ int shared_instance_points[];  // Shared memory to hold instance point calculations

        for (int instance_id = blockIdx.y; instance_id < nInstance; instance_id += gridDim.y) {
            int instance_total = instance_pointnum[instance_id];
            int intersection = 0;

            for (int i = start; i < end; i++) {
                int idx = proposals_idx[i];
                if ((int)instance_labels[idx] == instance_id) {
                    intersection += 1;
                }
            }

            // Calculate IOU and store in shared memory
            shared_instance_points[threadIdx.x] = intersection;
            __syncthreads();

            if(threadIdx.x == 0) {  // Single thread in the block computes final IOU
                int total_intersection = 0;
                for (int i = 0; i < blockDim.x; i++) {
                    total_intersection += shared_instance_points[i];
                }
                // Store result in global memory
                proposals_iou[proposal_id * nInstance + instance_id] = (float)total_intersection / ((float)(proposal_total + instance_total - total_intersection) + 1e-5);
            }
            __syncthreads();
        }
    }
}