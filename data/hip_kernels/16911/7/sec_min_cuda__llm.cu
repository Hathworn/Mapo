#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sec_min_cuda_(int nProposal, int C, float *inp, int *offsets, float *out) {
    // Use shared memory to store minimum values, one for each thread in a block
    extern __shared__ float shared_min[];
    int p_id = blockIdx.x;  // Each block processes one proposal

    if (p_id < nProposal) {
        int start = offsets[p_id];
        int end = offsets[p_id + 1];

        // Initialize shared memory
        for (int plane = threadIdx.x; plane < C; plane += blockDim.x) {
            shared_min[plane] = 1e50;
        }
        __syncthreads();  // Ensure all threads have initialized shared memory

        // Find minimum using shared memory
        for (int plane = threadIdx.x; plane < C; plane += blockDim.x) {
            float local_min = 1e50;
            for (int i = start; i < end; i++) {
                float val = inp[i * C + plane];
                if (val < local_min) {
                    local_min = val;
                }
            }
            // Reduce local_mins to shared memory for final min value
            atomicMin(&shared_min[plane], local_min);
        }
        __syncthreads();  // Ensure all threads have completed their atomic operations

        // Write result to global memory
        for (int plane = threadIdx.x; plane < C; plane += blockDim.x) {
            out[p_id * C + plane] = shared_min[plane];
        }
    }
}