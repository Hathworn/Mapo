#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sec_mean_cuda_(int nProposal, int C, float *inp, int *offsets, float *out) {
    // Using shared memory to store means for planes, reducing global memory accesses
    extern __shared__ float shared_means[];

    for (int p_id = blockIdx.x; p_id < nProposal; p_id += gridDim.x) {
        int start = offsets[p_id];
        int end = offsets[p_id + 1];
        float count = (float)(end - start);

        for (int plane = threadIdx.x; plane < C; plane += blockDim.x) {
            shared_means[threadIdx.x] = 0;
            __syncthreads(); // Barrier to ensure shared memory is ready

            for (int i = start; i < end; i++) {
                shared_means[threadIdx.x] += (inp[i * C + plane] / count);
            }
            
            __syncthreads(); // Barrier to ensure all threads have written to shared memory

            out[p_id * C + plane] = shared_means[threadIdx.x];
            __syncthreads(); // Final barrier to ensure syncing between iterations
        }
    }
}