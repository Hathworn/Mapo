#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sec_max_cuda_(int nProposal, int C, float *inp, int *offsets, float *out) {
    // Use shared memory to reduce global memory accesses
    extern __shared__ float shared_max[];

    for (int p_id = blockIdx.x; p_id < nProposal; p_id += gridDim.x) {
        int start = offsets[p_id];
        int end = offsets[p_id + 1];

        for (int plane = threadIdx.x; plane < C; plane += blockDim.x) {
            float max_val = -1e50;

            // Initialize shared memory for each thread
            shared_max[threadIdx.x] = -1e50;

            for (int i = start; i < end; i++) {
                float current_val = inp[i * C + plane];
                if (current_val > max_val) {
                    max_val = current_val;
                }
            }
            // Write each thread's maximum to shared memory
            shared_max[threadIdx.x] = max_val;

            // Synchronize threads to ensure shared memory is fully populated
            __syncthreads();

            // Reduce maximum across all threads in block
            if (threadIdx.x == 0) {
                float block_max = shared_max[0];
                for (int j = 1; j < blockDim.x; ++j) {
                    if (shared_max[j] > block_max) {
                        block_max = shared_max[j];
                    }
                }
                // Write the block's maximum to global memory
                out[p_id * C + plane] = block_max;
            }

            // Synchronize to ensure block maximum is written before next iteration
            __syncthreads();
        }
    }
}