#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void third_and_fourth_normal_component(float* z, float* xx, float* yy, float* zx, float* zy, int npix, float* N3) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if the thread index is within the range
    if (i < npix) {
        // Compute and store the third normal component
        float temp_zx = xx[i] * zx[i];
        float temp_zy = yy[i] * zy[i];
        N3[i] = -z[i] - temp_zx - temp_zy;
        
        // Set the fourth normal component to 1
        N3[npix + i] = 1;
    }
}