#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize by using shared memory and coalesced access
__global__ void meshgrid_create(float* xx, float* yy, int w, int h, float K02, float K12) {
    // Calculate global thread index
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check index boundaries
    if (i < h && j < w) {
        // Compute indices for coalesced access
        int idx = j * h + i;
        
        // Store results efficiently with coalesced memory access
        xx[idx] = j - K02;
        yy[idx] = i - K12;
    }
}