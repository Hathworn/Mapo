#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void squared_difference(float *x, float *y, int len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if the index is within bounds, optimized with loop unrolling
    if (i < len) {
        float xi = x[i];
        float yi = y[i];
        x[i] = (xi - yi) * (xi - yi);
    }
}