#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_A_ch_3(float* rho, float* dz, float* s_a, int npix, int nchannels, int nimages, float* A_ch) {
    // Optimize thread indexing by merging grid dimensions into a single index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalElements = npix * nimages * nchannels;
    
    // Reduce bounds check to a single comparison
    if (idx < totalElements) {
        int c = idx / (npix * nimages);
        int j = (idx / npix) % nimages;
        int i = idx % npix;
        A_ch[idx] = (rho[c * npix + i] / dz[i]) * s_a[c * nimages * 3 + j];
    }
}