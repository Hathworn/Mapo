#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_A_expansion(float* A, int* rowind, int* colind, float* val, int npix, int nimages) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int totalPixels = npix * nimages; // Precompute total pixels for boundary check
    
    if (i < totalPixels) { // Simplified boundary condition
        rowind[i] = i;
        colind[i] = i % npix;
        val[i] = A[i];
    }
}