#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_N(float* N, const float* norm, int npix_per_component) {
    // Calculate global index for components and pixels
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int c = blockIdx.y; // Use blockIdx.y directly for components

    // Process only valid pixel indices
    if (i < npix_per_component) {
        // Compute linear index once for better performance
        int index = c * npix_per_component + i;
        N[index] = N[index] / norm[i];
    }
}