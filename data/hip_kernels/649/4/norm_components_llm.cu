#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void norm_components(float* N, int npix, float* norm) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds
    if (i < npix) {
        // Load values into registers to reduce memory access
        float Nx = N[i];
        float Ny = N[npix + i];
        float Nz = N[npix * 2 + i];

        // Compute norm and store result
        norm[i] = fmaxf(1e-10, sqrtf(Nx * Nx + Ny * Ny + Nz * Nz));
    }
}