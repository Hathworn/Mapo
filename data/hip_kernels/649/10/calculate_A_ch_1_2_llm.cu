#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_A_ch_1_2(float* rho, float* dz, float* s_a, float* xx_or_yy, float* s_b, float K, int npix, int nchannels, int nimages, float* A_ch) {
    // Calculate global indices directly and combine logical checks
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < npix && j < nimages && c < nchannels) {
        // Pre-calculate repeated indices for efficiency
        int idx = c*npix*nimages + j*npix + i;
        int rho_idx = c*npix + i;
        int img_idx = c * nimages * 3 + j;

        // Compute the result with fewer arithmetic operations
        A_ch[idx] = (rho[rho_idx] / dz[i]) * (K * s_a[img_idx] - xx_or_yy[i] * s_b[img_idx]);
    }
}