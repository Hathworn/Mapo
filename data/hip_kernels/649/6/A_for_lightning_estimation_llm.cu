#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void A_for_lightning_estimation(float* rho, float* N, int npix, float* A) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize by removing redundant calculations
    int index = i + npix * (blockIdx.y + 4 * blockIdx.z); // Compute index outside the condition
    if (i < npix) {
        A[index] = rho[blockIdx.y * npix + i] * N[blockIdx.z * npix + i];  // Use precomputed index
    }
}