#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_B_for_depth(float* B, float* rho, float* Ns, int npix, int nchannels, int nimages) {
    // Optimize access using shared memory
    extern __shared__ float shared_rho[];
    
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int c = blockIdx.y*blockDim.y + threadIdx.y;
    
    if (c * npix + threadIdx.x < npix) {
        shared_rho[threadIdx.x] = rho[c*npix + threadIdx.x];
    }
    __syncthreads();

    if (i < npix * nimages) {
        B[c*npix*nimages + i] -= shared_rho[i % npix] * Ns[c*npix*nimages + i];
    }
}