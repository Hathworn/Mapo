#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_constant(int* arr, int k, int arr_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop unrolling for better performance on large arrays
    int stride = blockDim.x * gridDim.x;
    for (; i < arr_size; i += stride) {
        arr[i] += k;
    }
}