#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initialize_rho(float* rho, int size_c, int nc) {
    // Use a flattened 2D grid-to-thread mapping for better coalescing
    int index = blockIdx.y * size_c + blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size_c * nc) {
        rho[index] = 0.5f;
    }
}