#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_AT_expansion(float* A, int* rowind, int* colind, float* val, int npix, int nimages) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = npix * nimages; // Precompute total elements
    if (i < total_elements) {
        int image_index = i % nimages; // Calculate once
        int pixel_index = i / nimages; // Calculate once
        int col_index = pixel_index + image_index * npix; // Simplify col_index computation

        colind[i] = col_index; 
        rowind[i] = pixel_index;
        val[i] = A[col_index];
    }
}