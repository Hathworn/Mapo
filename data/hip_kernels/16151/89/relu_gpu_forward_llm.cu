#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;  // Calculate stride for loop unrolling
    for (int i = tid; i < N; i += stride) {  // Loop over data with stride
        out[i] = in[i] > 0 ? in[i] : 0;  // Apply ReLU in loop
    }
}