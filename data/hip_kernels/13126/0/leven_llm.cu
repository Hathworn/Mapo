#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void leven(char* a, char* b, char* costs, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i > 0 && i < size) {
        // Initialize first cost element for row
        costs[0] = i;
        int nw = i - 1;
        for(int j = 1; j <= size; j++) {
            // Use min() for clarity and readability
            int costReplace = (a[i - 1] == b[j - 1]) ? nw : nw + 1;
            int costInsertDelete = 1 + min(costs[j], costs[j-1]);
            int cj = min(costReplace, costInsertDelete);
            nw = costs[j];
            costs[j] = cj;
        }
    }
}