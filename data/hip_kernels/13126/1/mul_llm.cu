#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void mul(double* A, double* B, double* C, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // Calculate row index using 2D grid
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate column index using 2D grid

    if (row < size && col < size) {
        double sum = 0.0;
        // Perform dot product for A's row and B's column
        for (int k = 0; k < size; k++) {
            sum += A[row * size + k] * B[k * size + col];
        }
        C[row * size + col] = sum;  // Store result in C
    }
}