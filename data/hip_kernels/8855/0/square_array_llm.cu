#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to increase performance
    extern __shared__ float shared_a[];
    
    if (idx < N) {
        // Load data into shared memory
        shared_a[threadIdx.x] = a[idx];
        __syncthreads();

        // Perform computation
        shared_a[threadIdx.x] *= shared_a[threadIdx.x];
        __syncthreads();

        // Write results back to global memory
        a[idx] = shared_a[threadIdx.x];
    }
}