#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define HEIGHT 1024  // assuming HEIGHT is defined somewhere

__global__ void matrixAddPitch(int *a, int *b, int *c, int pitch) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Calculate x index
    int idy = threadIdx.y + blockIdx.y * blockDim.y;  // Calculate y index

    if (idx >= pitch || idy >= HEIGHT) return;  // Boundary check

    int index = idy * pitch + idx;  // Calculate 1D index from 2D coordinates
    c[index] = a[index] + b[index];  // Perform matrix addition
}