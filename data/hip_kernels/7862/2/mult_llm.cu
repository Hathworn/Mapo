#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int THREADS_PER_BLOCK = 32;
const int N = 2048;

__global__ void mult(int *a, int *b, int *c)
{
    int pos = threadIdx.x + blockDim.x * blockIdx.x;
    if (pos >= N) return;

    // Unroll loop for improved performance
    int gridSize = gridDim.x * blockDim.x;
    while (pos < N) {
        c[pos] = a[pos] * b[pos];
        pos += gridSize;
    }
}