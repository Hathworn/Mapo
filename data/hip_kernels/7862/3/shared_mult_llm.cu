#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
#define N 512

#define N 2048
#define THREADS_PER_BLOCK 512

*/
const int THREADS_PER_BLOCK = 32;
const int N = 2048;

__global__ void shared_mult(int *a, int *b, int *c)
{
    __shared__ int mem[THREADS_PER_BLOCK];
    int pos = threadIdx.x + blockIdx.x * blockDim.x;

    // Check if pos is within bounds
    if (pos < N) {
        mem[threadIdx.x] = a[pos] * b[pos];
    }

    __syncthreads();

    // Check if pos is within bounds before writing to output
    if (pos < N) {
        c[pos] = mem[threadIdx.x];
    }
}