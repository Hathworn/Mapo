#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int THREADS_PER_BLOCK = 32;
const int N = 2048;

__global__ void dotProd( int *a, int *b, int *c ) {
    __shared__ int temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure within bounds
    if (index < N) {
        temp[threadIdx.x] = a[index] * b[index];
    } else {
        temp[threadIdx.x] = 0;
    }
    __syncthreads(); // Wait for all threads to fill temp

    // Perform parallel reduction
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        __syncthreads();
        if (threadIdx.x < stride) {
            temp[threadIdx.x] += temp[threadIdx.x + stride];
        }
    }

    // Write the result for this block to global memory
    if (threadIdx.x == 0) {
        c[blockIdx.x] = temp[0];
    }
}