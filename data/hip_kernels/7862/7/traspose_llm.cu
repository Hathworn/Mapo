#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose(int *src, int *dest) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // Calculate column index
    int y = blockIdx.y * blockDim.y + threadIdx.y; // Calculate row index
    
    // Bounds checking
    if (x >= WIDTH || y >= HEIGHT) return;

    __shared__ int tile[TILE_DIM][TILE_DIM]; // Use shared memory for tiled access
    
    // Load data into shared memory tile
    tile[threadIdx.y][threadIdx.x] = src[y * WIDTH + x]; 
    __syncthreads(); // Synchronize threads to ensure complete data load

    // Write transposed data from shared memory
    int trans_x = blockIdx.y * blockDim.y + threadIdx.x; // Calculate destination column index
    int trans_y = blockIdx.x * blockDim.x + threadIdx.y; // Calculate destination row index
    
    if (trans_x < HEIGHT && trans_y < WIDTH) {
        dest[trans_y * HEIGHT + trans_x] = tile[threadIdx.x][threadIdx.y];
    }
}