#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define WIDTH 1024
#define HEIGHT 1024

__global__ void add(int* a, int* b, int* c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    // Use precomputed values for thread's global positions to avoid repetitive calculations
    int globalIdx = idy * WIDTH + idx;

    if (idx >= WIDTH || idy >= HEIGHT) return;

    // Perform addition using global index to improve memory access efficiency
    c[globalIdx] = a[globalIdx] + b[globalIdx];
}