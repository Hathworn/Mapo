#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int THREADS_PER_BLOCK = 32;
const int N = 2048;

__global__ void dotProd(int *a, int *b, int *c) {
    __shared__ int temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Each thread computes its product and stores in shared memory
    int t = (index < N) ? a[index] * b[index] : 0;

    // Reduce within a block
    temp[threadIdx.x] = t;
    __syncthreads();

    // Parallel reduction code (no loop unrolling for simplicity)
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            temp[threadIdx.x] += temp[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Sum results from blocks and write to global memory
    if (threadIdx.x == 0) {
        atomicAdd(c, temp[0]); // Use atomic operation to avoid race conditions
    }
}