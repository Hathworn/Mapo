#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy(int *src, int *dest)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Use `__syncthreads()` for synchronization
    __syncthreads();
    
    if (idx < WIDTH && idy < HEIGHT) 
    {
        // Coalesced memory access
        int index = idy * WIDTH + idx;
        dest[index] = src[index];
    }
}