#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelVacio( void ) {
    // Ensure only threads with threadIdx.x < 10 proceed
    if (threadIdx.x < 10) {
        // Using __syncthreads() for better synchronization control
        __syncthreads();
        printf("Data: %s Id Thread: %d Id block : %d Num threads block : %d\n", "helloWorld!", threadIdx.x, blockIdx.x, blockDim.x);
    }
}