#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define WIDTH 1024
#define HEIGHT 1024

__global__ void add(int* a, int* b, int* c)
{
    // Calculate global thread indices
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    // Boundary check
    if (idx < WIDTH && idy < HEIGHT)
    {
        // Perform element-wise addition
        int index = idy * WIDTH + idx;
        c[index] = a[index] + b[index];
    }
}