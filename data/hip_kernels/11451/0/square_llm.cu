#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(float * d_out, float * d_in) {
    // Use blockIdx and blockDim to handle larger input data
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure the index is within the bounds of the data
    if (idx < N) {
        float f = d_in[idx];
        d_out[idx] = f * f;
    }
}