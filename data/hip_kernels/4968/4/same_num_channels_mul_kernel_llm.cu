#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void same_num_channels_mul_kernel(const float *data_l, const float *data_r, float *result, int total)
{
    int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);

    // Use shared memory for improved memory access efficiency
    __shared__ float s_data_l[1024];
    __shared__ float s_data_r[1024];

    if (idx < 2 * total) {
        // Load data into shared memory
        s_data_l[threadIdx.x * 2] = data_l[idx];
        s_data_l[threadIdx.x * 2 + 1] = data_l[idx + 1];
        s_data_r[threadIdx.x * 2] = data_r[idx];
        s_data_r[threadIdx.x * 2 + 1] = data_r[idx + 1];

        __syncthreads(); // Ensure all data is loaded before processing

        // Compute results using shared memory
        result[idx] = s_data_l[threadIdx.x * 2] * s_data_r[threadIdx.x * 2] 
                    - s_data_l[threadIdx.x * 2 + 1] * s_data_r[threadIdx.x * 2 + 1];
        result[idx + 1] = s_data_l[threadIdx.x * 2] * s_data_r[threadIdx.x * 2 + 1]
                        + s_data_l[threadIdx.x * 2 + 1] * s_data_r[threadIdx.x * 2];
    }
}