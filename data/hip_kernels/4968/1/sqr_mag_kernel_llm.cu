#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sqr_mag_kernel(const float *data, float *result, int total)
{
    // Use a single index calculation, threads check their range safely
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int dataIdx = 2 * idx;

    if (idx < total) {
        float real = data[dataIdx];
        float imag = data[dataIdx + 1];
        result[idx] = real * real + imag * imag;
        result[dataIdx + 1] = 0.0f; // Simplified setting imaginary part to 0
    }
}