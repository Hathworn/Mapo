#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constant_add_kernel(const float *data_l, float constant, float *result, int total)
{
    // Use shared memory to minimize global memory access
    __shared__ float shared_data[512]; // Adjust size based on blockDim.x

    int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);

    // Load data to shared memory
    if (idx < 2 * total) {
        shared_data[threadIdx.x] = data_l[idx];
        if (idx + 1 < 2 * total) {
            shared_data[threadIdx.x + 1] = data_l[idx + 1];
        }
    }
    __syncthreads();

    // Perform computation using shared memory
    if (idx / 2 < total) {
        result[idx] = shared_data[threadIdx.x] + constant;
        if (idx + 1 < 2 * total) {
            result[idx + 1] = shared_data[threadIdx.x + 1];
        }
    }
}