#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void one_channel_mul_kernel(const float *data_l, const float *data_r, float *result, int channel_total, int total)
{
    // Calculate the global index for the current thread
    int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
    
    // Calculate one channel index
    int one_ch_idx = idx % (2 * channel_total);

    // Check if the current index is within the bounds
    if (idx / 2 < total) {
        // Compute the real and imaginary multiplication
        float real_temp = data_l[idx] * data_r[one_ch_idx] - data_l[idx + 1] * data_r[one_ch_idx + 1];
        float imag_temp = data_l[idx] * data_r[one_ch_idx + 1] + data_l[idx + 1] * data_r[one_ch_idx];
        
        // Store the results
        result[idx] = real_temp;
        result[idx + 1] = imag_temp;
    }
}