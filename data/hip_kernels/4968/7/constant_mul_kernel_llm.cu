#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constant_mul_kernel(const float *data_l, float constant, float *result, int total) 
{
    // Calculate unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Update loop condition to avoid unnecessary calculation
    if (idx < (total / 2)) {
        int data_idx = 2 * idx;
        
        // Perform vectorized operation for consecutive elements
        result[data_idx] = data_l[data_idx] * constant;
        result[data_idx + 1] = data_l[data_idx + 1] * constant;
    }
}