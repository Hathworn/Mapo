#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void sum_channels(float *dest, const float *src, uint channels, uint num_channel_elem)
{
    // Compute absolute thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds and return if index is outside
    if (idx >= num_channel_elem) return;
    
    // Use shared memory for performance improvement
    extern __shared__ float shared_src[];

    // Load data into shared memory
    for (uint i = threadIdx.x; i < channels * num_channel_elem; i += blockDim.x) {
        shared_src[i] = src[i];
    }

    // Synchronize to ensure shared memory is fully loaded
    __syncthreads();

    float acc = 0;

    // Accumulate values from shared memory
    for (uint i = 0; i < channels; ++i) {
        acc += shared_src[idx + i * num_channel_elem];
    }

    // Write accumulated result to destination
    dest[idx] = acc;
}
```
