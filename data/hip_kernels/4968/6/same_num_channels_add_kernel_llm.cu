#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void same_num_channels_add_kernel(const float *data_l, const float *data_r, float *result, int total)
{
    // Calculate global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize memory access by handling two elements per thread
    if (idx < total) {
        int address = 2 * idx;
        result[address] = data_l[address] + data_r[address];
        result[address + 1] = data_l[address + 1] + data_r[address + 1];
    }
}