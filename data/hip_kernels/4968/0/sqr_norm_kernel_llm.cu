#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sqr_norm_kernel(const float *in, float *block_res, int total)
{
    extern __shared__ float sdata[];
    int in_idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
    int i = threadIdx.x;
    unsigned ins = blockDim.x;

    // Initialize shared memory with input values squared
    sdata[i] = (in_idx < total * 2) ? (in[in_idx] * in[in_idx] + in[in_idx + 1] * in[in_idx + 1]) : 0;

    // Perform reduction using shared memory
    __syncthreads();
    for (unsigned stride = ins / 2; stride > 0; stride >>= 1) {
        if (i < stride && i + stride < ins) {
            sdata[i] += sdata[i + stride];
        }
        __syncthreads();
    }

    // Store the result in block_res
    if (i == 0) {
        block_res[blockIdx.x] = sdata[0];
    }
}