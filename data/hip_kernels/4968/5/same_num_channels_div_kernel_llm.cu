#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void same_num_channels_div_kernel(const float *data_l, const float *data_r, float *result, unsigned total)
{
    unsigned idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);

    // Check if the index is within bounds
    if (idx / 2 < total) {
        float real_l = data_l[idx];
        float imag_l = data_l[idx + 1];
        float real_r = data_r[idx];
        float imag_r = data_r[idx + 1];

        // Precalculate reused values
        float denom = real_r * real_r + imag_r * imag_r;
        float real = (real_l * real_r + imag_l * imag_r) / denom;
        float imag = (imag_l * real_r - real_l * imag_r) / denom;

        result[idx] = real;
        result[idx + 1] = imag;
    }
}