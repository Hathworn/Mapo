#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conj_kernel(const float *data, float *result, int total)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread processes a pair of elements
    int doubleIdx = 2 * idx;

    // Ensure index is within bounds
    if (idx < total) {
        // Copy and negate the imaginary part
        result[doubleIdx] = data[doubleIdx];
        result[doubleIdx + 1] = -data[doubleIdx + 1];
    }
}