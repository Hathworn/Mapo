#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(int* a, int* b, int* c, int n) {
    // Calculate the global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure valid access 
    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}