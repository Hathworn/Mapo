#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMul(int *a, int *b, int *c, int n) {
    // Calculate the row and column index of the element
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Establish a temporary sum for the current element
    int temp_sum = 0;

    // Ensure row and column are within bounds
    if (row < n && col < n) {
        // Iterate over the row and column to calculate the sum
        for (int k = 0; k < n; k++) {
            temp_sum += a[row * n + k] * b[k * n + col];
        }

        // Assign the calculated sum to the respective element in output matrix
        c[row * n + col] = temp_sum;
    }
}