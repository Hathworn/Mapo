#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(int* a, int* b, int* c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure tid is within bounds and process in a single if statement
    if (tid < n) c[tid] = a[tid] + b[tid];
}