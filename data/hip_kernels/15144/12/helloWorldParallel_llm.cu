#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloWorldParallel(void) {
    int i = threadIdx.x; 
    int j = blockIdx.x; 

    // Optimize printf usage by combining values into a single string for efficiency.
    printf("Hello world from GPU %d/%d\n", j, i);
}