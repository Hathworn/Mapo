#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void oddeven(int* x, int I, int n)
{
    // Calculate the global thread index for more flexibility
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for faster access within a block, allocate space accordingly
    extern __shared__ int shared[];
    
    // Load data from global memory to shared memory
    if (2 * id < n) {
        shared[2 * id] = x[2 * id];
        if (2 * id + 1 < n) {
            shared[2 * id + 1] = x[2 * id + 1];
        }
    }
    __syncthreads();

    // Perform swap for even phase
    if (I == 0 && 2 * id + 1 < n) {
        if (shared[2 * id] > shared[2 * id + 1]) {
            int temp = shared[2 * id];
            shared[2 * id] = shared[2 * id + 1];
            shared[2 * id + 1] = temp;
        }
    }
    // Perform swap for odd phase
    if (I == 1 && 2 * id + 2 < n) {
        if (shared[2 * id + 1] > shared[2 * id + 2]) {
            int temp = shared[2 * id + 1];
            shared[2 * id + 1] = shared[2 * id + 2];
            shared[2 * id + 2] = temp;
        }
    }
    __syncthreads();

    // Store data back to global memory
    if (2 * id < n) {
        x[2 * id] = shared[2 * id];
        if (2 * id + 1 < n) {
            x[2 * id + 1] = shared[2 * id + 1];
        }
    }
}