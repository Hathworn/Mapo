#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
{
}

__global__ void addKernel(int* c, const int* a, const int* b)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform addition if within bounds (Assumes array size >= gridDim.x * blockDim.x)
    c[i] = a[i] + b[i];
}