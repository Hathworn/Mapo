#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Compute the bottom left corner of the target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];

    // Precompute frequently used terms
    float dxdy = dx * dy;
    float dx1 = 1.0f - dx;
    float dy1 = 1.0f - dy;
    
    // Fill pixel containing bottom right corner
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        float weight = dxdy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);  // Use atomicAdd directly for efficiency
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Fill pixel containing bottom left corner
    tx -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        float weight = dx1 * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Fill pixel containing upper left corner
    ty -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        float weight = dx1 * dy1;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Fill pixel containing upper right corner
    tx += 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        float weight = dx * dy1;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}