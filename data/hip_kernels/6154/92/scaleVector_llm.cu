#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len) 
{
    // Calculate global thread position
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure position is within vector bounds
    if (pos < len) 
    {
        d_res[pos] = d_src[pos] * scale; // Scale and assign
    }
}