#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Load loc into register to reduce global memory access
        short2 loc = loc_[ptidx];

        // Perform scaled assignment
        float scaled_x = loc.x * scale;
        float scaled_y = loc.y * scale;

        // Write back results to global memory 
        x[ptidx] = scaled_x;
        y[ptidx] = scaled_y;
    }
}