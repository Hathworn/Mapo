#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Calculate global thread index in x and y directions
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Early exit for threads out of image bounds
    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Calculate bottom left corner position using time_scale
    float cx = fmaf(u_, time_scale, (float)j + 1.0f); // Using fmaf for fused-multiply-add
    float cy = fmaf(v_, time_scale, (float)i + 1.0f); // Using fmaf for fused-multiply-add

    // Convert bottom left corner position to nearest integer pixel index
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    float value = src[image_row_offset + j];
    // Ensure the pixel falls within bounds before atomic addition
    if ((tx >= 0) && (tx < w) && (ty >= 0) && (ty < h))
    {
        // Using atomic add to avoid race conditions
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}