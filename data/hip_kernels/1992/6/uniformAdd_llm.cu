#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) {
    __shared__ float uni;
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];

    // Use blockDim.x for efficient calculation of address
    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x;

    __syncthreads();
    
    // Use local variable to avoid redundant global memory access
    float local_uni = uni;

    // Perform adds more efficiently
    g_data[address] += local_uni;
    if (threadIdx.x + blockDim.x < n)
        g_data[address + blockDim.x] += local_uni;
}