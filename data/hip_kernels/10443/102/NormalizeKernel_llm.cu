#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global thread indices
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundary conditions for early exit
    if (i < h && j < w)
    {
        const int pos = i * s + j;

        // Load scale once from global memory
        float scale = normalization_factor[pos];

        // Use ternary operator for conditional scaling
        float invScale = (scale != 0.0f) ? (1.0f / scale) : 1.0f;

        // Multiply the image value by invScale
        image[pos] *= invScale;
    }
}