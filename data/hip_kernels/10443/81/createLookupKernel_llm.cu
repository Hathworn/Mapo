#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void createLookupKernel(const int* inds, int total, int* output)
{
    // Calculate global thread index once
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce global memory access if needed
    if (idx < total)
    {
        int index = inds[idx];
        output[index] = idx;
    }
}