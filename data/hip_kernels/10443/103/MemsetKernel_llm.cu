#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate the linear index directly
    int pos = blockIdx.x * blockDim.x + threadIdx.x + 
              (blockIdx.y * blockDim.y + threadIdx.y) * w;

    if (pos >= w * h) return; // Exit if out of bounds

    image[pos] = value; // Assign value
}