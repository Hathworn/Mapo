#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tissueGPU3Kernel(float *d_tissxyz, float *d_vessxyz, float *d_pt000, float *d_qv000, int nnt, int nnv, int is2d, float req, float r2d) 
{
    int itp = blockDim.x * blockIdx.x + threadIdx.x;
    int nnv2 = 2 * nnv;
    float req2 = req * req;
    float r2d2 = r2d * r2d;
    
    if (itp < nnt) {
        float xt = d_tissxyz[itp];
        float yt = d_tissxyz[itp + nnt];
        float zt = d_tissxyz[itp + nnt * 2];
        float p = 0.0f;

        #pragma unroll // Unroll loop for improved performance
        for (int jvp = 0; jvp < nnv; jvp++) {
            float x = d_vessxyz[jvp] - xt;
            float y = d_vessxyz[jvp + nnv] - yt;
            float z = d_vessxyz[jvp + nnv2] - zt;
            float dist2 = x * x + y * y + z * z;
            
            float gtv;
            if (dist2 < req2) {
                if (is2d) gtv = log(r2d2 / req2) + 1.0f - dist2 / req2;
                else gtv = (1.5f - 0.5f * dist2 / req2) / req;
            } else {
                if (is2d) gtv = log(r2d2 / dist2);
                else gtv = 1.0f / sqrt(dist2);
            }
            p += d_qv000[jvp] * gtv;
        }
        d_pt000[itp] = p;
    }
}