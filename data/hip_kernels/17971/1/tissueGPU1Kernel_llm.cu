#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/***********************************************************
tissueGPU1.cu
GPU kernel to accumulate contributions of tissue source
strengths qt to tissue solute levels pt.
TWS December 2011
Cuda 10.1 Version, August 2019
************************************************************/

__global__ void tissueGPU1Kernel(int *d_tisspoints, float *d_dtt000, float *d_pt000, float *d_qt000, int nnt)
{
    int itp = blockDim.x * blockIdx.x + threadIdx.x;
    int nnt2 = 2 * nnt;
    float p = 0.0f;

    if (itp < nnt) {
        int ix = d_tisspoints[itp];
        int iy = d_tisspoints[itp + nnt];
        int iz = d_tisspoints[itp + nnt2];

        // Use shared memory to optimize access to d_tisspoints
        __shared__ int s_tisspoints[1024]; // assuming a max block size, tune appropriately

        // Load shared memory
        if (threadIdx.x < nnt) {
            s_tisspoints[threadIdx.x] = d_tisspoints[threadIdx.x];
            s_tisspoints[threadIdx.x + nnt] = d_tisspoints[threadIdx.x + nnt];
            s_tisspoints[threadIdx.x + nnt2] = d_tisspoints[threadIdx.x + nnt2];
        }
        __syncthreads();

        for (int jtp = 0; jtp < nnt; jtp++) {
            // Access shared memory instead of global memory
            int jx = s_tisspoints[jtp];
            int jy = s_tisspoints[jtp + nnt];
            int jz = s_tisspoints[jtp + nnt2];
            int ixyz = abs(jx - ix) + abs(jy - iy) + abs(jz - iz);
            p += d_qt000[jtp] * d_dtt000[ixyz];
        }
        d_pt000[itp] = p;
    }
}