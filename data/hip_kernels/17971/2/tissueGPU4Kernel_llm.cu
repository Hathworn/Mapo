#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tissueGPU4Kernel(int *d_tisspoints, float *d_dtt000, float *d_qtp000, float *d_xt, float *d_rt, int nnt, int step, float diff)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int itp = i / step;
    int itp1 = i % step;
    int nnt2 = 2 * nnt;
    float r = 0.0f;

    if(itp < nnt){
        int ix = d_tisspoints[itp];
        int iy = d_tisspoints[itp + nnt];
        int iz = d_tisspoints[itp + nnt2];

        for(int jtp = itp1; jtp < nnt; jtp += step){
            int ixyz = abs(d_tisspoints[jtp] - ix) + abs(d_tisspoints[jtp + nnt] - iy) + abs(d_tisspoints[jtp + nnt2] - iz);
            r -= d_dtt000[ixyz] * d_qtp000[jtp] * d_xt[jtp];
        }

        r /= diff;
        r += d_xt[itp];  // diagonal of matrix has 1s

        if(itp1 == 0) d_rt[itp] = r;

        // Use a single loop with proper condition to ensure d_rt update
        for(int istep = 1; istep < step; istep++){
            __syncthreads();
            if(itp1 == istep) d_rt[itp] += r;
        }
    }
}