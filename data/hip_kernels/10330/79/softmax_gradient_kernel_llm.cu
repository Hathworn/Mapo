#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    
    float tmp = 0;

    // Load data into registers and reduce across warp using warp reduce
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Use warp reduction to minimize synchronization overhead
    if (idx < 32) { // Assume blockDim.x is a multiple of warp size
        for (int i = idx + 32; i < blockDim.x; i += 32) {
            reduction_buffer[idx] += reduction_buffer[i];
        }
    }
    __syncthreads(); // Ensure warp reduction is complete

    if (idx == 0) {
        for (int i = 1; i < 32; ++i) {
            reduction_buffer[0] += reduction_buffer[i];
        }
    }
    __syncthreads(); // Ensure reduction computation is complete

    // Compute gradient using the reduced sum
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}