#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Ensure single-thread execution using block and thread index check
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        *accuracy /= N;
    }
}