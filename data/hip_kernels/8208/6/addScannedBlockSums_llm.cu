#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addScannedBlockSums(float *input, float *aux, int len) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int dx = blockDim.x;
    int i = 2 * bx * dx + tx;

    // Pre-calculate previous block sum to avoid repeated computation
    float prevBlockSum = (bx > 0) ? input[bx - 1] : 0;

    if (i < len) {
        aux[i] += prevBlockSum; // Add block sum to element i
    }

    if (i + dx < len) {
        aux[i + dx] += prevBlockSum; // Add block sum to element i + dx
    }
}