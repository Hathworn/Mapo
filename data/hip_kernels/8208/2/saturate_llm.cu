#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saturate(unsigned int *bins, unsigned int num_bins) {

    // Use thread index to determine starting bin index, iterate through all relevant bins for each thread
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Loop through all bins that the thread is responsible for
    for (int i = index; i < num_bins; i += stride) {
        if (bins[i] >= 128) {
            bins[i] = 127; // Clamp value at 127
        }
    }
}