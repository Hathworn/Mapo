#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__global__ void matrixMultiply(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    // Calculate row and column indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float pValue = 0;

    // Ensure within bounds
    if (row < numCRows && col < numCColumns) {
        // Accumulate result
        for (int k = 0; k < numAColumns; k++) {
            pValue += A[row * numAColumns + k] * B[k * numBColumns + col];
        }
        // Write to output matrix
        C[row * numCColumns + col] = pValue;
    }
}