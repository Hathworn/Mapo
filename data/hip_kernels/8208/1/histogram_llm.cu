#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogram(unsigned int *input, unsigned int *bins, unsigned int num_elements, unsigned int num_bins) {

    // Use shared memory for thread-private histograms
    __shared__ unsigned int hist[NUM_BINS];

    int threadId = threadIdx.x;
    int globalId = blockDim.x * blockIdx.x + threadIdx.x;
    int elementsPerThread = (num_bins + blockDim.x - 1) / blockDim.x; // Adjust for efficient memory use

    // Initialize histogram bins to zero
    for (int j = 0; j < elementsPerThread; ++j) {
        int histIdx = threadId + j * blockDim.x;
        if (histIdx < num_bins) {
            hist[histIdx] = 0;
        }
    }
    __syncthreads();

    // Build histogram privately in shared memory
    if (globalId < num_elements) {
        atomicAdd(&hist[input[globalId]], 1);
    }
    __syncthreads();

    // Combine private histograms into final global histogram
    for (int k = 0; k < elementsPerThread; ++k) {
        int binIdx = threadId + k * blockDim.x;
        if (binIdx < num_bins) {
            atomicAdd(&bins[binIdx], hist[binIdx]);
        }
    }
}