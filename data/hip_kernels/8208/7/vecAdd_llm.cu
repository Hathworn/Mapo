#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundary condition and add vectors
    if (i < len) {
        out[i] = in1[i] + in2[i];
    }
}