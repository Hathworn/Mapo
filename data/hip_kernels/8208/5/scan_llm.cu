#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan(float *input, float *output, float *aux, int len) {

    // Declare shared memory
    __shared__ float XY[2 * BLOCK_SIZE];

    // Calculate global and local thread index
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int i = 2 * bx * blockDim.x + tx;

    // Load data into shared memory with bounds checking
    XY[tx] = (i < len) ? input[i] : 0.0f;
    XY[tx + blockDim.x] = (i + blockDim.x < len) ? input[i + blockDim.x] : 0.0f;

    // Ensure all threads have loaded their data
    __syncthreads();

    // Up-sweep phase
    for (unsigned int stride = 1; stride <= BLOCK_SIZE; stride *= 2) {
        __syncthreads();
        int index = (tx + 1) * stride * 2 - 1;
        if (index < 2 * BLOCK_SIZE)
            XY[index] += XY[index - stride];
    }

    // Down-sweep phase
    for (int stride = BLOCK_SIZE / 2; stride > 0; stride /= 2) {
        __syncthreads();
        int index = (tx + 1) * stride * 2 - 1;
        if (index + stride < 2 * BLOCK_SIZE)
            XY[index + stride] += XY[index];
    }

    // Synchronize before writing data back to global memory
    __syncthreads();

    // Write data back to global memory with bounds checking
    if (i < len)
        output[i] = XY[tx];
    if (i + blockDim.x < len)
        output[i + blockDim.x] = XY[tx + blockDim.x];

    // Store block sum to aux array
    if (aux != NULL && tx == 0)
        aux[bx] = XY[2 * blockDim.x - 1];
}