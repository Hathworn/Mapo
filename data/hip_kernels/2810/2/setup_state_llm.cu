#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Launch with one thread as each thread initializes a single state
    int idx = threadIdx.x;  
    hiprand_init(seed, idx, 0, &state[idx]);  // Utilize thread index for state initialization
}