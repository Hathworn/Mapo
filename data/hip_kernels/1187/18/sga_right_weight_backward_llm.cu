#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sga_right_weight_backward(const int n, const float *bottom_data, const float *top_data, const float *temp_diff, const int height, const int width, const int depth, const int wsize, float *filters_diff) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) {
        return;
    }
    int step = height * width;
    int base = index / step * step * depth + index % step;
    int fbase = index / step * step * wsize + index % step;
    int col = index % step % width;

    // Use registers to reduce memory access latency
    const float *temp_diff_base = temp_diff + base;
    const float *bottom_data_base = bottom_data + base;
    for (int i = 0; i < depth; ++i) {
        filters_diff[fbase] += temp_diff_base[i * step] * bottom_data_base[i * step];
    }

    if (col - 1 >= 0) {
        int location = fbase + step;
        const float *top_data_offset1 = top_data + base - 1;
        for (int i = 0; i < depth; ++i) {
            filters_diff[location] += temp_diff_base[i * step] * top_data_offset1[i * step];
        }

        location = fbase + 3 * step;
        filters_diff[location] += temp_diff_base[0] * bottom_data_base[0];
        for (int i = 1; i < depth; ++i) {
            filters_diff[location] += temp_diff_base[i * step] * top_data_offset1[(i - 1) * step];
        }

        location = fbase + 4 * step;
        filters_diff[location] += temp_diff_base[(depth - 1) * step] * bottom_data_base[(depth - 1) * step];
        for (int i = 0; i < depth - 1; ++i) {
            filters_diff[location] += temp_diff_base[i * step] * top_data_offset1[(i + 1) * step];
        }
    }

    if (col - 2 >= 0) {
        int location = fbase + 2 * step;
        const float *top_data_offset2 = top_data + base - 2;
        for (int i = 0; i < depth; ++i) {
            filters_diff[location] += temp_diff_base[i * step] * top_data_offset2[i * step];
        }
    }
}