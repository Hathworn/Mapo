#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sga_up_forward (const int n, const float *filters, const int height, const int width, const int depth, const int wsize, float *top_data){

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index >= n) {
        return;
    }

    int step = height * width;
    int base = (index / width) * step * depth + index % width;
    int fbase = (index / width) * step * wsize + index % width;

    for (int row = height - 1; row >= 0; row--) {
        int shift = fbase + row * width;
        for (int d = 0; d < depth; d++) {
            float temp = 0;
            int location = base + d * step + row * width;

            float top_val = top_data[location];
            float filter_shift_step = filters[shift + step];
            float filter_shift_2step = filters[shift + 2 * step];
            float filter_shift_3step = filters[shift + 3 * step];
            float filter_shift_4step = filters[shift + 4 * step];
            
            // Load current data point
            temp += top_val * filters[shift];

            // Optimize boundary check by using conditional operator
            temp += ((row + 1 < height) ? top_data[location + width] : top_val) * filter_shift_step;
            temp += ((row + 2 < height) ? top_data[location + 2 * width] : top_val) * filter_shift_2step;
            temp += ((row + 1 < height && d - 1 >= 0) ? top_data[location + width - step] : top_val) * filter_shift_3step;
            temp += ((row + 1 < height && d + 1 < depth) ? top_data[location + width + step] : top_val) * filter_shift_4step;

            // Store computed result back
            top_data[location] = temp;
        }
    }
}