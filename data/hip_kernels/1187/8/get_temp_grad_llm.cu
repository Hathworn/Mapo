#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void get_temp_grad(const int n, const float *gradOutput, const float *mask, float *top_grad, const int mask_index) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) { // Reverse condition logic to reduce branching
        if (((int)mask[index]) == mask_index) {
            top_grad[index] = gradOutput[index];
        }
    }
}