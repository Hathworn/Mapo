#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sga_left_data_backward (const int n, const float *filters, float *top_diff, const int height, const int width, const int depth, const int wsize, float *bottom_diff){

    // Calculate the global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if the index is out of bounds
    if (index >= n) {
        return;
    }

    // Calculate steps and bases
    int step = height * width;
    int base = index / height * step * depth + (index % height) * width;
    int fbase = index / height * step * wsize + (index % height) * width;

    // Iterate over each column
    for (int col = 0; col < width; col++) {
        int shift = fbase + col;

        // Iterate over each depth level
        for (int d = 0; d < depth; d++) {
            int location = base + d * step + col;
            float temp = top_diff[location];

            if (col - 1 >= 0)
                temp += top_diff[location - 1] * filters[shift - 1 + step];
            if (col - 2 >= 0)
                temp += top_diff[location - 2] * filters[shift - 2 + 2 * step];
            if (col - 1 >= 0 && d + 1 < depth)
                temp += top_diff[location - 1 + step] * filters[shift - 1 + 3 * step];
            if (col - 1 >= 0 && d - 1 >= 0)
                temp += top_diff[location - 1 - step] * filters[shift - 1 + 4 * step];

            // Update top_diff with the computed temp
            top_diff[location] = temp;

            // Compute and accumulate bottom_diff
            bottom_diff[location] += temp * filters[shift];
        }
    }

    // Post processing: iterate over each column again
    for (int col = 0; col < width; col++) {
        int shift = fbase + col;
        int location = base + col;
        bottom_diff[location] += top_diff[location] * filters[shift + 3 * step];
        location += (depth - 1) * step;
        bottom_diff[location] += top_diff[location] * filters[shift + 4 * step];
    }
}