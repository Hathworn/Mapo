#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sga_down_weight_backward(const int n, const float *bottom_data, const float *top_data, const float *temp_diff, const int height, const int width, const int depth, const int wsize, float *filters_diff) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) {
        return;
    }

    int step = height * width;
    int base = index / step * step * depth + index % step;
    int fbase = index / step * step * wsize + index % step;
    int row = index % step / width;

    // Unroll loop to improve performance
    for (int i = 0; i < depth; i++) {
        int offset = base + i * step;
        float tmp_diff_val = temp_diff[offset];
        filters_diff[fbase] += tmp_diff_val * bottom_data[offset];
    }

    if (row - 1 >= 0) {
        int location_step = fbase + step;
        for (int i = 0; i < depth; i++) {
            int offset = base + i * step;
            filters_diff[location_step] += temp_diff[offset] * top_data[offset - width];
        }

        int location_3step = fbase + 3 * step;
        filters_diff[location_3step] += temp_diff[base] * bottom_data[base];
        for (int i = 1; i < depth; i++) {
            int offset = base + i * step;
            filters_diff[location_3step] += temp_diff[offset] * top_data[base + (i - 1) * step - width];
        }

        int location_4step = fbase + 4 * step;
        int last_offset = base + (depth - 1) * step;
        filters_diff[location_4step] += temp_diff[last_offset] * bottom_data[last_offset];
        for (int i = 0; i < depth - 1; i++) {
            int offset = base + i * step;
            filters_diff[location_4step] += temp_diff[offset] * top_data[base + (i + 1) * step - width];
        }
    }

    if (row - 2 >= 0) {
        int location_2step = fbase + 2 * step;
        for (int i = 0; i < depth; i++) {
            int offset = base + i * step;
            filters_diff[location_2step] += temp_diff[offset] * top_data[offset - 2 * width];
        }
    }
}