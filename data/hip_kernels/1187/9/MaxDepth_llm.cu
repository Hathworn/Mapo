#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MaxDepth(const int n, const float *bottom_data, const int step, const int depth, float *idx) {
    // Calculate the unique thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds
    if (index >= n) return;

    int base = index / step * step * depth + index % step;
    float max_val = bottom_data[base];
    int max_k = 0;

    // Unroll the loop to improve performance
    for (int i = 1; i < depth; ++i) {
        float val = bottom_data[base + i * step];
        if (val > max_val) {
            max_val = val;
            max_k = i;
        }
    }
    // Store the result in the output index
    idx[index] = max_k;
}