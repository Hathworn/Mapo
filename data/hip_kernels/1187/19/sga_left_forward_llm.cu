#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sga_left_forward(const int n, const float *filters, const int height, const int width, const int depth, const int wsize, float *top_data)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= n)
    {
        return;
    }
    
    int step = height * width;
    int base = index / height * step * depth + (index % height) * width;
    int fbase = index / height * step * wsize + (index % height) * width;

    for (int col = width - 1; col >= 0; col--)
    {
        int shift = fbase + col;
        for (int d = 0; d < depth; d++)
        {
            int location = base + d * step + col;
            float temp = 0;

            temp += top_data[location] * filters[shift];
            if (col + 1 < width)
            {
                temp += top_data[location + 1] * filters[shift + step];
                if (col + 2 < width)
                {
                    temp += top_data[location + 2] * filters[shift + 2 * step];
                }
                else
                {
                    temp += top_data[location] * filters[shift + 2 * step];
                }
                
                if (d - 1 >= 0)
                {
                    temp += top_data[location + 1 - step] * filters[shift + 3 * step];
                }
                else
                {
                    temp += top_data[location] * filters[shift + 3 * step];
                }
                
                if (d + 1 < depth)
                {
                    temp += top_data[location + 1 + step] * filters[shift + 4 * step];
                }
                else
                {
                    temp += top_data[location] * filters[shift + 4 * step];
                }
            }
            else
            {
                temp += top_data[location] * filters[shift + step]
                     + top_data[location] * filters[shift + 2 * step]
                     + top_data[location] * filters[shift + 3 * step]
                     + top_data[location] * filters[shift + 4 * step];
            }

            top_data[location] = temp;
        }
    }
}