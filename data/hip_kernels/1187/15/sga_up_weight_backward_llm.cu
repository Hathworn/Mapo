#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sga_up_weight_backward (const int n, const float *bottom_data, const float *top_data, const float *temp_diff, const int height, const int width, const int depth, const int wsize, float *filters_diff) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) {
        return;
    }

    int step = height * width;
    int base = index / step * step * depth + index % step;
    int fbase = index / step * step * wsize + index % step;
    int row = index % step / width;

    // Use shared memory for faster access patterns
    __shared__ float local_temp_diff[1024]; 
    __shared__ float local_bottom_data[1024];
    __shared__ float local_top_data[1024];

    if (threadIdx.x < step * depth) {
        local_temp_diff[threadIdx.x] = temp_diff[threadIdx.x];
        local_bottom_data[threadIdx.x] = bottom_data[threadIdx.x];
        local_top_data[threadIdx.x] = top_data[threadIdx.x];
    }
    __syncthreads();

    for (int i = 0; i < depth; i++) {
        filters_diff[fbase] += local_temp_diff[base + i * step] * local_bottom_data[base + i * step];
    }

    // Minimize index operation by pre-storing recurrent calculations
    if (row + 1 < height) {
        int location = fbase + step;
        for (int i = 0; i < depth; i++) {
            float temp_val = local_temp_diff[base + i * step];
            filters_diff[location] += temp_val * local_top_data[base + i * step + width];
            filters_diff[fbase + 3 * step] += temp_val * (i == 0 ? local_bottom_data[base] : local_top_data[base + (i - 1) * step + width]);
            filters_diff[fbase + 4 * step] += (i == depth - 1 ? temp_val * local_bottom_data[base + i * step] : temp_val * local_top_data[base + (i + 1) * step + width]);
        }
    }

    if (row + 2 < height) {
        int location = fbase + 2 * step;
        for (int i = 0; i < depth; i++) {
            filters_diff[location] += local_temp_diff[base + i * step] * local_top_data[base + i * step + 2 * width];
        }
    }
}