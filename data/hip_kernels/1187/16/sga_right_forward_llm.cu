#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sga_right_forward(const int n, const float *filters, const int height, const int width, const int depth, const int wsize, float *top_data){

    // Calculate the index of the thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (index >= n) {
        return;
    }

    int step = height * width;
    int base = index / height * step * depth + (index % height) * width;
    int fbase = index / height * step * wsize + (index % height) * width;

    // Loop over columns
    for (int col = 0; col < width; col++) {
        int shift = fbase + col;

        // Loop over depth
        for (int d = 0; d < depth; d++) {
            float temp = 0;
            int location = base + d * step + col;

            // Sum up the contributions
            temp += top_data[location] * filters[shift];
            temp += top_data[col - 1 >= 0 ? location - 1 : location] * filters[shift + step];
            temp += top_data[col - 2 >= 0 ? location - 2 : location] * filters[shift + 2 * step];
            temp += top_data[(col - 1 >= 0 && d - 1 >= 0) ? location - 1 - step : location] * filters[shift + 3 * step];
            temp += top_data[(col - 1 >= 0 && d + 1 < depth) ? location - 1 + step : location] * filters[shift + 4 * step];

            // Store the result
            top_data[location] = temp;
        }
    }
}