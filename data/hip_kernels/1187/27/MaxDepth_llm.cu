#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MaxDepth (const int n, const float *bottom_data, const int step, const int depth, float *idx){

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n)
    {
        return;
    }
    
    int base = index / step * step * depth + index % step;
    int k = 0;
    float max_val = bottom_data[base];

    // Use unrolling and shared memory to improve performance
    for (int i = 1; i < depth; i++)
    {
        float current_val = bottom_data[base + i * step];
        if (current_val > max_val)
        {
            max_val = current_val;
            k = i;
        }
    }
    
    idx[index] = k;
}