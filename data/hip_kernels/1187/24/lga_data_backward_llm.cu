#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lga_data_backward(const int n, const float *filters, const float *top_diff, const int height, const int width, const int channel, const int radius, float *bottom_diff) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) return;

    int step = height * width;
    int wsize = 2 * radius + 1;
    int fbase = index / (step * channel) * (step * wsize * wsize * 3) + index % step;
    int row = (index % step) / width;
    int col = index % width;
    int depth = (index / step) % channel;

    // Precompute constant values to reduce redundant calculations
    int topShiftBase = index - (row * width + col + depth * step);

    for (int d = -1; d <= 1; d++) {
        for (int r = -radius; r <= radius; r++) {
            for (int c = -radius; c <= radius; c++) {
                int rr = r + row;
                int cc = c + col;
                int dd = d + depth;
                if (rr >= 0 && cc >= 0 && dd >= 0 && rr < height && cc < width && dd < channel) {
                    int shift = r * width + c + d * step;
                    int location = (-d + 1) * (wsize * wsize) + (-r + radius) * wsize - c + radius;
                    bottom_diff[index] += top_diff[topShiftBase + shift] * filters[fbase + r * width + c + location * step];
                } else {
                    int location = (d + 1) * (wsize * wsize) + (r + radius) * wsize + c + radius;
                    bottom_diff[index] += top_diff[topShiftBase] * filters[fbase + location * step];
                }
            }
        }
    }
}