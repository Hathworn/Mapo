#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sga_left_weight_backward(const int n, const float *bottom_data, const float *top_data, const float *temp_diff, const int height, const int width, const int depth, const int wsize, float *filters_diff) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) return;

    int step = height * width;
    int base = index / step * step * depth + index % step; // up->down
    int fbase = index / step * step * wsize + index % step;
    int col = index % step % width;

    // Optimize loop: reduce repeated pointer arithmetic
    int base_step;

    for (int i = 0; i < depth; i++) {
        base_step = base + i * step;
        filters_diff[fbase] += temp_diff[base_step] * bottom_data[base_step];
    }
    
    if (col + 1 < width) {
        int location = fbase + step;
        for (int i = 0; i < depth; i++) {
            base_step = base + i * step;
            filters_diff[location] += temp_diff[base_step] * top_data[base_step + 1];
        }
        
        location = fbase + 3 * step;
        filters_diff[location] += temp_diff[base] * bottom_data[base];
        for (int i = 1; i < depth; i++) {
            base_step = base + i * step;
            filters_diff[location] += temp_diff[base_step] * top_data[base + (i - 1) * step + 1];
        }
        
        location = fbase + 4 * step;
        filters_diff[location] += temp_diff[base + (depth - 1) * step] * bottom_data[base + (depth - 1) * step];
        for (int i = 0; i < depth - 1; i++) {
            base_step = base + i * step;
            filters_diff[location] += temp_diff[base_step] * top_data[base + (i + 1) * step + 1];
        }
    }

    if (col + 2 < width) {
        int location = fbase + 2 * step;
        for (int i = 0; i < depth; i++) {
            base_step = base + i * step;
            filters_diff[location] += temp_diff[base_step] * top_data[base_step + 2];
        }
    }
}