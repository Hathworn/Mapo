#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void get_temp_grad(const int n, const float *gradOutput, const float *mask, float *top_grad, const int mask_index) {
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within the bounds
    if (index < n) {
        // Use mask matching condition to determine grad assignment
        if (__float_as_int(mask[index]) == mask_index) {
            top_grad[index] = gradOutput[index];
        }
    }
}