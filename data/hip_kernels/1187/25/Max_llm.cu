#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Max(const int n, const float *top_temp, float *top_data, float *mask, const int mask_index) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) { // Combine boundary check with index usage
        float temp_val = top_temp[index]; // Cache memory access
        float top_val = top_data[index];
        if (top_val < temp_val) {
            top_data[index] = temp_val;
            mask[index] = mask_index;
        }
    }
}