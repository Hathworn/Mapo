#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sga_right_data_backward (const int n, const float *filters, float *top_diff, const int height, const int width, const int depth, const int wsize, float *bottom_diff){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) {
        return;
    }
    
    int step = height * width;
    int base = index / height * step * depth + (index % height) * width;
    int fbase = index / height * step * wsize + (index % height) * width;

    #pragma unroll
    for (int col = width - 1; col >= 0; col--) {
        int shift = fbase + col;
        #pragma unroll
        for (int d = 0; d < depth; d++) {
            int location = base + d * step + col;
            float temp = top_diff[location];
            if (col + 1 < width)
                temp += top_diff[location + 1] * filters[shift + 1 + step];
            if (col + 2 < width)
                temp += top_diff[location + 2] * filters[shift + 2 + 2 * step];
            if (col + 1 < width && d + 1 < depth)
                temp += top_diff[location + 1 + step] * filters[shift + 1 + 3 * step];
            if (col + 1 < width && d - 1 >= 0)
                temp += top_diff[location + 1 - step] * filters[shift + 1 + 4 * step];
            top_diff[location] = temp;
            bottom_diff[location] += (temp * filters[shift]);
        }
    }

    #pragma unroll
    for (int col = 0; col < width; col++) {
        int shift = fbase + col;
        int location = base + col;
        bottom_diff[location] += top_diff[location] * filters[shift + 3 * step];
        location += (depth - 1) * step;
        bottom_diff[location] += top_diff[location] * filters[shift + 4 * step];
    }
}