#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void lga_filter_backward (const int n, const float *bottom_data, const float *top_diff, const int height, const int width, const int channel, const int radius, float *filter_diff) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < n) { // Combine condition to avoid unnecessary computation
        int step = height * width;
        int wsize = 2 * radius + 1;

        int base = index / (step * wsize * wsize * 3) * (step * channel) + index % step;
        int location = index / step % (wsize * wsize * 3);
        int d = location / (wsize * wsize) - 1;
        int r = (location / wsize) % wsize - radius;
        int c = location % wsize - radius;

        int rr = index % step / width + r;
        int cc = index % width + c;

        for (int i = 0; i < channel; i++) {
            int dd = i + d;
            int idx_shift = base + i * step; // Precompute repeated index calculation

            if (rr >= 0 && cc >= 0 && dd >= 0 && rr < height && cc < width && dd < channel) {
                int shift = r * width + c + d * step;
                filter_diff[index] += top_diff[idx_shift] * bottom_data[base + shift + i * step];
            } else {
                filter_diff[index] += top_diff[idx_shift] * bottom_data[idx_shift];
            }
        }
    }
}