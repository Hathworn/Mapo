#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lga_filtering_forward(const int n, const float *bottom_data, const float *filters, const int height, const int width, const int channel, const int radius, float *top_data) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within valid range
    if (index >= n) {
        return;
    }

    // Calculate necessary variables and offsets
    int step = height * width;
    int wsize = 2 * radius + 1;
    int fbase = index / (step * channel) * (step * wsize * wsize * 3) + index % step;
    int row = index % step / width;
    int col = index % width;
    int depth = index / step % channel;

    // Loop unrolling for efficiency
    #pragma unroll
    for (int d = -1; d <= 1; d++) {
        int dd = d + depth;
        if (dd < 0 || dd >= channel) continue;
        
        for (int r = -radius; r <= radius; r++) {
            int rr = r + row;
            if (rr < 0 || rr >= height) continue;

            for (int c = -radius; c <= radius; c++) {
                int cc = c + col;
                if (cc < 0 || cc >= width) continue;

                int shift = r * width + c + d * step;
                int location = (d + 1) * (wsize * wsize) + (r + radius) * wsize + c + radius;
                top_data[index] += bottom_data[index + shift] * filters[fbase + location * step];
            }
        }
    }
}