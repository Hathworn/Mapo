#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Max(const int n, const float *top_temp, float *top_data, float *mask, const int mask_index) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n && top_data[index] < top_temp[index]) { // Combine conditions for efficiency
        top_data[index] = top_temp[index];
        mask[index] = mask_index; 
    }
}