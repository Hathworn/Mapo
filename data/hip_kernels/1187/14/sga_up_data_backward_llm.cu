#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sga_up_data_backward (const int n, const float *filters, float *top_diff, const int height, const int width, const int depth, const int wsize, float *bottom_diff) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) {
        return;
    }
    int step = height * width;
    int base = index / width * step * depth + index % width; // up->down
    int fbase = index / width * step * wsize + index % width;

    // Using shared memory for filters to reduce global memory accesses
    extern __shared__ float s_filters[];
    for (int row = threadIdx.y; row < height; row += blockDim.y) {
        int shift = fbase + row * width;
        s_filters[threadIdx.y * width + threadIdx.x] = filters[shift];
        __syncthreads();

        for (int d = 0; d < depth; d++) {
            int location = base + d * step + row * width;
            float temp = top_diff[location];
            if (row - 1 >= 0)
                temp += top_diff[location - width] * s_filters[threadIdx.y * width + threadIdx.x - width + step];
            if (row - 2 >= 0)
                temp += top_diff[location - 2 * width] * s_filters[threadIdx.y * width + threadIdx.x - 2 * width + 2 * step];
            if (row - 1 >= 0 && d + 1 < depth)
                temp += top_diff[location - width + step] * s_filters[threadIdx.y * width + threadIdx.x - width + 3 * step];
            if (row - 1 >= 0 && d - 1 >= 0)
                temp += top_diff[location - width - step] * s_filters[threadIdx.y * width + threadIdx.x - width + 4 * step];
            top_diff[location] = temp;
            bottom_diff[location] += temp * s_filters[threadIdx.y * width + threadIdx.x];
        }
        __syncthreads();
    }

    for (int row = 0; row < height; row++) {
        int shift = fbase + row * width;
        int location = base + row * width;
        bottom_diff[location] += top_diff[location] * filters[shift + 3 * step];
        location += (depth - 1) * step;
        bottom_diff[location] += top_diff[location] * filters[shift + 4 * step];
    }
}