#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sga_down_forward(const int n, const float *filters, const int height, const int width, const int depth, const int wsize, float *top_data) {
    // Compute global index and check bounds
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) return;

    // Compute common variables
    int step = height * width;
    int base = (index / width) * step * depth + (index % width); // up->down
    int fbase = (index / width) * step * wsize + (index % width);

    // Iterate through each row
    for (int row = 0; row < height; row++) {
        int shift = fbase + row * width;

        // Iterate through each depth
        for (int d = 0; d < depth; d++) {
            float temp = 0;
            int location = base + d * step + row * width;

            // Apply filter logic using ternary operators for conciseness
            temp += top_data[location] * filters[shift];
            temp += (row - 1 >= 0 ? top_data[location - width] : top_data[location]) * filters[shift + step];
            temp += (row - 2 >= 0 ? top_data[location - 2 * width] : top_data[location]) * filters[shift + 2 * step];
            temp += (row - 1 >= 0 && d - 1 >= 0 ? top_data[location - width - step] : top_data[location]) * filters[shift + 3 * step];
            temp += (row - 1 >= 0 && d + 1 < depth ? top_data[location - width + step] : top_data[location]) * filters[shift + 4 * step];

            // Store the result back
            top_data[location] = temp;
        }
    }
}