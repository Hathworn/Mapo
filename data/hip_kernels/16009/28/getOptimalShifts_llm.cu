#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getOptimalShifts( float2 * __restrict__ optimalShifts, const float2 * __restrict__ bestShifts, int imageCount, int tileCountX, int tileCountY, int optimalShiftsPitch, int referenceImage, int imageToTrack)
{
    // Calculate global thread index
    int tileIdxX = blockIdx.x * blockDim.x + threadIdx.x;
    int tileIdxY = blockIdx.y * blockDim.y + threadIdx.y;

    // Avoid checking conditions inside loop, early exit if out of bounds
    if (tileIdxX >= tileCountX || tileIdxY >= tileCountY) return;

    int n1 = imageCount - 1;

    // Pre-compute pointer to the current tile's shifts
    const float2* r = &bestShifts[(tileIdxX + tileIdxY * tileCountX) * n1];

    // Use a single loop by adjusting the sign of the range accordingly
    float2 totalShift = make_float2(0, 0);
    int start = min(referenceImage, imageToTrack);
    int end = max(referenceImage, imageToTrack);
    int sign = (referenceImage < imageToTrack) ? 1 : -1;

    for (int i = start; i < end; i++)
    {
        totalShift.x += sign * r[i].x;
        totalShift.y += sign * r[i].y;
    }

    // Efficiently calculate the position for storing the total shift using pointer arithmetic
    *((float2*)((char*)(optimalShifts) + optimalShiftsPitch * tileIdxY) + tileIdxX) = totalShift;
}