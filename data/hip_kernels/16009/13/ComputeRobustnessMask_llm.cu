#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeRobustnessMask( const float3* __restrict__ rawImgRef, const float3* __restrict__ rawImgMoved, float4* __restrict__ robustnessMask, hipTextureObject_t texUV, int imgWidth, int imgHeight, int imgPitch, int maskPitch, float alpha, float beta, float thresholdM)
{
    int pxX = blockIdx.x * blockDim.x + threadIdx.x;
    int pxY = blockIdx.y * blockDim.y + threadIdx.y;

    if (pxX >= imgWidth - 1 || pxY >= imgHeight - 1 || pxX < 1 || pxY < 1)
        return;

    extern __shared__ float3 pixelsRef[];
    int sharedOffset = 3 * (threadIdx.y * blockDim.x + threadIdx.x);

    float3 meanRef = make_float3(0.0f);
    float3 meanMoved = make_float3(0.0f);
    float3 stdRef = make_float3(0.0f);
    float3 dist = make_float3(0.0f);
    float3 sigma = make_float3(0.0f);

    float2 shiftf = tex2D<float2>(texUV, ((float)pxX + 0.5f) / (float)imgWidth, ((float)pxY + 0.5f) / (float)imgHeight);
    float2 maxShift = shiftf;
    float2 minShift = shiftf;

    // Calculate max and min shifts
    #pragma unroll
    for (int y = -2; y <= 2; y++)
    {
        for (int x = -2; x <= 2; x++)
        {
            float2 s = tex2D<float2>(texUV, ((float)pxX + x + 0.5f) / (float)imgWidth, ((float)pxY + y + 0.5f) / (float)imgHeight);
            maxShift.x = fmaxf(s.x, maxShift.x);
            maxShift.y = fmaxf(s.y, maxShift.y);
            minShift.x = fminf(s.x, minShift.x);
            minShift.y = fminf(s.y, minShift.y);
        }
    }

    int2 shift;
    shift.x = roundf(shiftf.x * 0.5f);
    shift.y = roundf(shiftf.y * 0.5f);

    // Calculate means
    #pragma unroll
    for (int y = -1; y <= 1; y++)
    {
        for (int x = -1; x <= 1; x++)
        {
            float3 refPixel = *((float3*)((char*)rawImgRef + imgPitch * (pxY + y)) + pxX + x);
            pixelsRef[sharedOffset + (y + 1) * 3 + (x + 1)] = refPixel;

            meanRef += refPixel;

            int ppy = min(max(pxY + shift.y + y, 0), imgHeight - 1);
            int ppx = min(max(pxX + shift.x + x, 0), imgWidth - 1);
            float3 movedPixel = *((float3*)((char*)rawImgMoved + imgPitch * ppy) + ppx);
            meanMoved += movedPixel;
        }
    }

    meanRef *= 1.0f / 9.0f;
    meanMoved *= 1.0f / 9.0f;

    float meandist = (fabs(meanRef.x - meanMoved.x) + fabs(meanRef.y - meanMoved.y) + fabs(meanRef.z - meanMoved.z)) / 3.0f;
    maxShift *= 0.5f * meandist;
    minShift *= 0.5f * meandist;

    float M = hypotf(maxShift.x - minShift.x, maxShift.y - minShift.y);

    // Calculate standard deviation
    #pragma unroll
    for (int y = -1; y <= 1; y++)
    {
        for (int x = -1; x <= 1; x++)
        {
            int p = sharedOffset + (y + 1) * 3 + (x + 1);
            float3 diff = pixelsRef[p] - meanRef;
            stdRef += diff * diff;
        }
    }

    stdRef = sqrtf(stdRef * (1.0f / 9.0f));

    float3 sigmaMD;
    sigmaMD.x = sqrtf(alpha * meanRef.x + beta);
    sigmaMD.y = sqrtf(alpha * meanRef.y + beta) * M_SQRT1_2; // watch for two green pixels averaged
    sigmaMD.z = sqrtf(alpha * meanRef.z + beta);

    dist = fabs(meanRef - meanMoved);

    sigma.x = fmaxf(sigmaMD.x, stdRef.x);
    sigma.y = fmaxf(sigmaMD.y, stdRef.y);
    sigma.z = fmaxf(sigmaMD.z, stdRef.z);

    dist.x *= (stdRef.x * stdRef.x / (stdRef.x * stdRef.x + sigmaMD.x * sigmaMD.x));
    dist.y *= (stdRef.y * stdRef.y / (stdRef.y * stdRef.y + sigmaMD.y * sigmaMD.y));
    dist.z *= (stdRef.z * stdRef.z / (stdRef.z * stdRef.z + sigmaMD.z * sigmaMD.z));

    float4 mask;
    float s = (M > thresholdM) ? 0.0f : 1.5f;

    const float t = 0.12f;
    mask.x = fmaxf(fminf(s * expf(-dist.x * dist.x / (sigma.x * sigma.x)) - t, 1.0f), 0.0f);
    mask.y = fmaxf(fminf(s * expf(-dist.y * dist.y / (sigma.y * sigma.y)) - t, 1.0f), 0.0f);
    mask.z = fmaxf(fminf(s * expf(-dist.z * dist.z / (sigma.z * sigma.z)) - t, 1.0f), 0.0f);
    mask.w = M;

    *(((float4*)((char*)robustnessMask + maskPitch * pxY)) + pxX) = mask;
}