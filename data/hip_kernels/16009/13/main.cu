#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "ComputeRobustnessMask.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    const float3 __restrict__ *rawImgRef = NULL;
hipMalloc(&rawImgRef, XSIZE*YSIZE);
const float3 __restrict__ *rawImgMoved = NULL;
hipMalloc(&rawImgMoved, XSIZE*YSIZE);
float4 __restrict__ *robustnessMask = NULL;
hipMalloc(&robustnessMask, XSIZE*YSIZE);
hipTextureObject_t texUV = 1;
int imgWidth = XSIZE;
int imgHeight = YSIZE;
int imgPitch = 2;
int maskPitch = 2;
float alpha = 2;
float beta = 2;
float thresholdM = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
ComputeRobustnessMask<<<gridBlock, threadBlock>>>(rawImgRef,rawImgMoved,robustnessMask,texUV,imgWidth,imgHeight,imgPitch,maskPitch,alpha,beta,thresholdM);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
ComputeRobustnessMask<<<gridBlock, threadBlock>>>(rawImgRef,rawImgMoved,robustnessMask,texUV,imgWidth,imgHeight,imgPitch,maskPitch,alpha,beta,thresholdM);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
ComputeRobustnessMask<<<gridBlock, threadBlock>>>(rawImgRef,rawImgMoved,robustnessMask,texUV,imgWidth,imgHeight,imgPitch,maskPitch,alpha,beta,thresholdM);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}