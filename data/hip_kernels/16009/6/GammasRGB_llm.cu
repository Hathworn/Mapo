#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float applysRGBGamma(float valIn)
{
    if (valIn <= 0.0031308f)
    {
        return 12.92f * valIn;
    }
    else
    {
        return (1.0f + 0.055f) * powf(valIn, 1.0f / 2.4f) - 0.055f;
    }
}

__global__ void GammasRGB(float3* __restrict__ inOutImg, int imgWidth, int imgHeight, int imgPitch)
{
    int pxX = blockIdx.x * blockDim.x + threadIdx.x;
    int pxY = blockIdx.y * blockDim.y + threadIdx.y;

    // Early exit for out-of-bounds pixels
    if (pxX >= imgWidth || pxY >= imgHeight) return;

    // Load and process pixel value
    float3* pixelPtr = (float3*)((char*)inOutImg + imgPitch * pxY) + pxX;
    float3 val = *pixelPtr;

    // Apply gamma correction with NaN handling and clamping
    val.x = applysRGBGamma(fmaxf(fminf(isnan(val.x) ? 0 : val.x, 1.0f), 0.0f));
    val.y = applysRGBGamma(fmaxf(fminf(isnan(val.y) ? 0 : val.y, 1.0f), 0.0f));
    val.z = applysRGBGamma(fmaxf(fminf(isnan(val.z) ? 0 : val.z, 1.0f), 0.0f));

    // Store the result
    *pixelPtr = val;
}