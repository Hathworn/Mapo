#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ApplyWeighting( float3 * __restrict__ inOutImg, const float3 * __restrict__ finalImg, const float3 * __restrict__ weight, int imgWidth, int imgHeight, int imgPitch, float threshold)
{
    int pxX = blockIdx.x * blockDim.x + threadIdx.x;
    int pxY = blockIdx.y * blockDim.y + threadIdx.y;

    // Early exit for threads out of bounds
    if (pxX >= imgWidth || pxY >= imgHeight)
        return;

    float3* inOutImgRow = (float3*)((char*)inOutImg + imgPitch * pxY);
    float3* finalImgRow = (float3*)((char*)finalImg + imgPitch * pxY);
    float3* weightRow = (float3*)((char*)weight + imgPitch * pxY);

    float3 inout = inOutImgRow[pxX];
    float3 val = finalImgRow[pxX];
    float3 w = weightRow[pxX];

    // Process each component
    for (int i = 0; i < 3; ++i) {
        if (reinterpret_cast<float*>(&w)[i] < threshold) {
            reinterpret_cast<float*>(&val)[i] += reinterpret_cast<float*>(&inout)[i];
            reinterpret_cast<float*>(&w)[i] += 1;
        }
        reinterpret_cast<float*>(&inout)[i] = 0;
        if (reinterpret_cast<float*>(&w)[i] != 0) {
            reinterpret_cast<float*>(&inout)[i] = reinterpret_cast<float*>(&val)[i] / reinterpret_cast<float*>(&w)[i];
        }
    }

    inOutImgRow[pxX] = inout;
}