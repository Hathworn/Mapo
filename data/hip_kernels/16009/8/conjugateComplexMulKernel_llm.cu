#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//Squared sum of a tile without the border
extern "C"

//Boxfilter ignoring the border parts
//blockDim.X must be tileSize + 2 * maxShift
//blockDim.Y must be 1
extern "C"
//Boxfilter ignoring the border parts
//blockDim.Y must be tileSize + 2 * maxShift
//blockDim.X must be 1
extern "C"

//Computed the normalized CC values out of the different input data
//Cross correlation is fft shifted
//blockDim.X must be 2 * maxShift
//blockDim.Y must be 2 * maxShift
//blockDim.Z must be nr of tiles
extern "C"

//Convert a tiled image into consecutive tiles for FFT
//input img has a pitch, output tiles are consecutive
//output tiles overlap by maxShift is filled by zero
extern "C"

//Convert a tiled image into consecutive tiles for FFT
//input img has a pitch, output tiles are consecutive
//output tiles overlap by maxShift on each side
extern "C"

__global__ void conjugateComplexMulKernel(const float2* __restrict__ aIn, float2* __restrict__ bInOut, int maxElem)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Early exit to minimize unnecessary computation
    if (idx >= maxElem) return;

    // Efficiently fetch the inputs once
    float2 valA = aIn[idx];
    float2 valB = bInOut[idx];

    // Precomputed conjugate complex
    float valA_y_conj = -valA.y;

    // Perform complex multiplication
    float2 res;
    res.x = valA.x * valB.x - valA_y_conj * valB.y;
    res.y = valA.x * valB.y + valA_y_conj * valB.x;
    
    // Store the result back
    bInOut[idx] = res;
}