#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeKernelParam( float3* __restrict__ kernelImg, int imgWidth, int imgHeight, int imgOutPitch, float Dth, float Dtr, float kDetail, float kDenoise, float kStretch, float kShrink)
{
    int pxX = blockIdx.x * blockDim.x + threadIdx.x;
    int pxY = blockIdx.y * blockDim.y + threadIdx.y;

    if (pxX >= imgWidth || pxY >= imgHeight)
        return;

    float3 grad = *(((float3*)((char*)kernelImg + imgOutPitch * pxY)) + pxX);
    float a11 = grad.x;
    float a22 = grad.y;
    float a12 = grad.z;

    // Optimized to avoid recalculating repeated expressions
    float a22_a11 = a22 - a11;
    float help = sqrtf(a22_a11 * a22_a11 + 4.0f * a12 * a12);

    float c = 2.0f * a12;
    float s = a22_a11 + help;

    float norm = sqrtf(c * c + s * s);
    if (norm > 0)
    {
        c /= norm;
        s /= norm;
    }
    else
    {
        c = 1;
        s = 0;
    }

    float lam1 = (a11 + a22 + help) * 0.5f;
    float lam2 = (a11 + a22 - help) * 0.5f;

    // Precompute constants to minimize operations
    float lam1_lam2 = lam1 + lam2;
    float A = 1 + sqrtf((lam1 - lam2) * (lam1 - lam2) / (lam1_lam2 * lam1_lam2));
    float D = 1 - sqrtf(lam1) / Dtr + Dth;
    D = fmaxf(fminf(1.0f, D), 0.0f);

    float A_kDetail = kDetail * A;
    float A_kDetail_kDenoise = kDetail * kDenoise;
    float k1h = kStretch * A_kDetail;
    float k2h = A_kDetail / kShrink;

    float k1 = ((1.0f - D) * k1h + D * A_kDetail_kDenoise);
    float k2 = ((1.0f - D) * k2h + D * A_kDetail_kDenoise);
    k1 *= k1; // Square terms for final calculation
    k2 *= k2;

    // Using intermediate variables to minimize computation
    float x1s = s;
    float x2s = c;
    float y1s = -c;
    float y2s = s;

    float b11 = k1 * x1s * x1s + x2s * x2s * k2;
    float b12 = k1 * x1s * y1s + x2s * y2s * k2;
    float b22 = k1 * y1s * y1s + y2s * y2s * k2;

    float det = b11 * b22 - b12 * b12 + 0.0000000001f;
    float3 kernel;
    kernel.x = b22 / det;
    kernel.y = b11 / det;
    kernel.z = -b12 / det;
    *(((float3*)((char*)kernelImg + imgOutPitch * pxY)) + pxX) = kernel;
}