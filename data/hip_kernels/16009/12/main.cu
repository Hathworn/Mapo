#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "ComputeKernelParam.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float3 __restrict__ *kernelImg = NULL;
hipMalloc(&kernelImg, XSIZE*YSIZE);
int imgWidth = XSIZE;
int imgHeight = YSIZE;
int imgOutPitch = 2;
float Dth = 1;
float Dtr = 1;
float kDetail = 1;
float kDenoise = 1;
float kStretch = 1;
float kShrink = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
ComputeKernelParam<<<gridBlock, threadBlock>>>(kernelImg,imgWidth,imgHeight,imgOutPitch,Dth,Dtr,kDetail,kDenoise,kStretch,kShrink);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
ComputeKernelParam<<<gridBlock, threadBlock>>>(kernelImg,imgWidth,imgHeight,imgOutPitch,Dth,Dtr,kDetail,kDenoise,kStretch,kShrink);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
ComputeKernelParam<<<gridBlock, threadBlock>>>(kernelImg,imgWidth,imgHeight,imgOutPitch,Dth,Dtr,kDetail,kDenoise,kStretch,kShrink);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}