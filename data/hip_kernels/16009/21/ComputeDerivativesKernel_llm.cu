#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeDerivativesKernel(int width, int height, int stride, float* Ix, float* Iy, float* Iz, hipTextureObject_t texSource, hipTextureObject_t texTarget)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix >= width || iy >= height) return;

    float dx = 1.0f / (float)width;
    float dy = 1.0f / (float)height;

    float x = ((float)ix + 0.5f) * dx;
    float y = ((float)iy + 0.5f) * dy;

    // Precompute texture coordinates to reduce redundant calculations
    float x_dx2 = x + 2.0f * dx;
    float x_dx1 = x + 1.0f * dx;
    float x_ndx1 = x - 1.0f * dx;
    float x_ndx2 = x - 2.0f * dx;

    float y_dy2 = y + 2.0f * dy;
    float y_dy1 = y + 1.0f * dy;
    float y_ndy1 = y - 1.0f * dy;
    float y_ndy2 = y - 2.0f * dy;

    float t0, t1;
    
    // x derivative
    t0 = tex2D<float>(texSource, x_dx2, y);
    t1 = tex2D<float>(texTarget, x_dx2, y);
    t0 -= tex2D<float>(texSource, x_dx1, y) * 8.0f;
    t1 -= tex2D<float>(texTarget, x_dx1, y) * 8.0f;
    t0 += tex2D<float>(texSource, x_ndx1, y) * 8.0f;
    t1 += tex2D<float>(texTarget, x_ndx1, y) * 8.0f;
    t0 -= tex2D<float>(texSource, x_ndx2, y);
    t1 -= tex2D<float>(texTarget, x_ndx2, y);
    t0 /= 12.0f;
    t1 /= 12.0f;
    
    // Store x derivative result
    *(((float*)((char*)Ix + stride * iy)) + ix) = (t0 + t1) * 0.5f;

    // t derivative
    *(((float*)((char*)Iz + stride * iy)) + ix) = tex2D<float>(texSource, x, y) - tex2D<float>(texTarget, x, y);

    // y derivative
    t0 = tex2D<float>(texSource, x, y_dy2);
    t1 = tex2D<float>(texTarget, x, y_dy2);
    t0 -= tex2D<float>(texSource, x, y_dy1) * 8.0f;
    t1 -= tex2D<float>(texTarget, x, y_dy1) * 8.0f;
    t0 += tex2D<float>(texSource, x, y_ndy1) * 8.0f;
    t1 += tex2D<float>(texTarget, x, y_ndy1) * 8.0f;
    t0 -= tex2D<float>(texSource, x, y_ndy2);
    t1 -= tex2D<float>(texTarget, x, y_ndy2);
    t0 /= 12.0f;
    t1 /= 12.0f;
    
    // Store y derivative result
    *(((float*)((char*)Iy + stride * iy)) + ix) = (t0 + t1) * 0.5f;
}