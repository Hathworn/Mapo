#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void patchmatch_r_argmax_kernel(float *conv, float *target, float *match, int *correspondence, int c1, int h1, int w1, int h2, int w2)
{
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int size1 = h1 * w1, size2 = h2 * w2;

    if (id1 >= size1) return;  // Early exit if thread out of bounds

    double conv_max = -1e20;
    int best_x2 = -1, best_y2 = -1;

    for (int id2 = 0; id2 < size2; id2++) {  // Loop unrolling over y2 and x2
        float conv_result = conv[id1 * size2 + id2];

        if (conv_result > conv_max) {
            conv_max = conv_result;
            best_x2 = id2 % w2;  // Compute x2
            best_y2 = id2 / w2;  // Compute y2
        }
    }
    
    if (conv_max > -1e20) {  // Update correspondence and match only if a valid max was found
        correspondence[id1 * 2 + 0] = best_x2;
        correspondence[id1 * 2 + 1] = best_y2;
        for (int c = 0; c < c1; c++) {
            match[c * size1 + id1] = target[c * size2 + best_y2 * w2 + best_x2];
        }
    }
}