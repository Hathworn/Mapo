#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))


__global__ void blend_kernel(float *A, float *BP, float *M, float *AP, float alpha, int c, int h, int w) {
    int _id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;

    // Check if thread index is within bounds
    if (_id < c * size) {
        int id = _id % size;
        int dc = _id / size;
        float weight = (M[id] < 0.05f) ? 0.f : alpha;

        // Perform blending operations
        AP[_id] = A[_id] * weight + BP[_id] * (1.f - weight);
    }
}