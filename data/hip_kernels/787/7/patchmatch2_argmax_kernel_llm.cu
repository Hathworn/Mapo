#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void patchmatch2_argmax_kernel(float *conv, int *prev_corrAB_upsampled, int *corrAB, int s_rad, int c, int h, int w)
{
    int h1 = h, w1 = w;
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int size1 = h1 * w1;
    int s_size = 2 * s_rad + 1;

    if (id1 < size1) {
        float conv_max = -1;

        int x2 = prev_corrAB_upsampled[2 * id1 + 0];
        int y2 = prev_corrAB_upsampled[2 * id1 + 1];

        // Use register variables for loop limit calculations
        int x2_min = MAX(0, x2 - s_rad);
        int x2_max = MIN(w1 - 1, x2 + s_rad);
        int y2_min = MAX(0, y2 - s_rad);
        int y2_max = MIN(h1 - 1, y2 + s_rad);

        for (int dx2 = x2_min - x2; dx2 <= x2_max - x2; dx2++) {
            for (int dy2 = y2_min - y2; dy2 <= y2_max - y2; dy2++) {
                int new_x2 = x2 + dx2;
                int new_y2 = y2 + dy2;
                int s_idx = (dy2 + s_rad) * s_size + (dx2 + s_rad);
                int id = id1 * s_size * s_size + s_idx;
                float conv_result = conv[id];

                if (conv_result > conv_max) {
                    conv_max = conv_result;
                    corrAB[id1 * 2 + 0] = new_x2;
                    corrAB[id1 * 2 + 1] = new_y2;
                }
            }
        }
    }

    return;
}