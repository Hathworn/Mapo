#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    // Use warp-level operations for reduction within warps
    float tmp = 0.0f;
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }

    // Perform warp-wise reduction
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        tmp += __shfl_down_sync(0xFFFFFFFF, tmp, offset);
    }

    // Reduction among warps using shared memory
    if (idx % warpSize == 0) {
        reduction_buffer[idx / warpSize] = tmp;
    }
    __syncthreads();

    // Final reduction within shared memory
    if (idx < (blockDim.x / warpSize)) {
        tmp = reduction_buffer[idx];
        for (int i = (blockDim.x / warpSize) + idx; i < SOFTMAX_NUM_THREADS; i += blockDim.x / warpSize) {
            tmp += reduction_buffer[i];
        }
    }
    __syncthreads();

    if (idx < warpSize) {
        if (idx == 0) {
            reduction_buffer[0] = tmp;
        }
    }
    __syncthreads();

    // Compute gradient
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}