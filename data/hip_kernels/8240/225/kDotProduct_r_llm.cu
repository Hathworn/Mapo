#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;

    // Optimization: Use warp-synchronous mechanism for efficiency
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        shmem[threadIdx.x] += a[eidx] * b[eidx];
    }
    __syncthreads();

    if (threadIdx.x < 32) {
        // Unroll final accumulation to fit within a warp
        volatile float* mysh = &shmem[threadIdx.x];
        mysh[0] += mysh[32] + mysh[16] + mysh[8] + mysh[4] + mysh[2] + mysh[1];
        
        if (threadIdx.x == 0) {
            target[blockIdx.x] = mysh[0];
        }
    }
}