#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate the unique global thread index
    const uint tidx = blockIdx.x * blockDim.x + threadIdx.x;
    // Initialize hiprand state with unique seed and sequence
    hiprand_init(seed, tidx, 0, &state[tidx]);
}