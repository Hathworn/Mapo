#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x; // Rename numThreads to stride for clarity
    
    // Precompute total elements in target for easy comparison in loop
    const unsigned int numEls = tgtWidth * tgtHeight; 
    
    for (uint i = idx; i < numEls; i += stride) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        
        // Efficient memory access using precomputed indexes
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}