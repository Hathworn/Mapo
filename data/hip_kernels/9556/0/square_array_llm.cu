#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    // Use shared memory for caching data
    extern __shared__ float cache[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < N) {
        // Load from global to shared memory
        cache[threadIdx.x] = a[idx];
        __syncthreads();
        
        // Work on shared memory
        cache[threadIdx.x] *= cache[threadIdx.x];
        
        // Store back to global memory
        __syncthreads();
        a[idx] = cache[threadIdx.x];
    }
}