#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_random_seed(unsigned int seed, hiprandState_t *d_curand_state) {
    int neuron = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid-stride loop to ensure scalability and handle larger problem sizes
    for (int i = neuron; i < NEURON_COUNT; i += gridDim.x * blockDim.x) {
        hiprand_init(seed, i, 0, &d_curand_state[i]);
    }
}