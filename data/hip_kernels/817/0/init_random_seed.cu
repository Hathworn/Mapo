#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_random_seed(unsigned int seed, hiprandState_t *d_curand_state) {
int neuron = blockIdx.x*blockDim.x + threadIdx.x;
hiprand_init(seed, neuron, 0, &d_curand_state[neuron]);
}