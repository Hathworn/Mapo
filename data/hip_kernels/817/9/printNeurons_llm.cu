#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void printNeurons(int *d_neuron_outputs, float *d_weighted_sums){
    // Calculate global thread index
    int neuron = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if the index is within bounds
    if (neuron < gridDim.x * blockDim.x) {
        printf("neuron: %d, weighted sum: %.2f, output: %d\n", neuron, d_weighted_sums[neuron], d_neuron_outputs[neuron]);
    }
}