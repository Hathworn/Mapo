#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use threadIdx.x == 0 to ensure only one thread performs the division
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}