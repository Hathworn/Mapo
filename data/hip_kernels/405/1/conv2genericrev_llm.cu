#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv2genericrev(float *input, float *kernel, float *output, int input_n, int input_h, int input_w, int kernel_n, int kernel_h, int kernel_w, float alpha, int stride_h, int stride_w)
{
    // output dimensions
    int output_h = input_h - (kernel_h - 1) * stride_h;
    int output_w = input_w - (kernel_w - 1) * stride_w;
    
    // this thread only processes one output, defined by the block Ids
    int kk = blockIdx.x;
    int ii = blockIdx.y;
    
    // batch id
    int batch = threadIdx.z;
    
    // kernel id
    int kid = threadIdx.x;
    int nkids = blockDim.x;
    
    // thread ID
    int tid = kid + batch * blockDim.x;
    int nthreads = blockDim.x * blockDim.z;
    
    // Pointer update for the output
    output = output + (kk * input_n + ii) * output_h * output_w;
    
    // Shared memory for output
    __shared__ float shared_output[CUDA_SHARED_MEM_SIZE];
    
    // Initialize shared memory array
    float *output_s = shared_output + tid * output_w * output_h;
    
    // Convolution loop
    float *input_p, *kernel_p, *output_p;
    int xx, yy, kx, ky;
    yy = threadIdx.y;
    output_p = output_s + yy * output_w;
    for(xx = 0; xx < output_w; xx++) {
        // Calculate dot product between input image and kernel
        input_p = input + (ii + batch * input_n) * input_h * input_w + yy * stride_h * input_w + xx * stride_w;
        kernel_p = kernel + (kk + batch * kernel_n) * kernel_w * kernel_h;
        float sum = 0;
        
        // Loop for kernel height
        for(ky = 0; ky < kernel_h; ky++) {
            // Loop for kernel width with step equal to nkids
            for(kx = kid; kx < kernel_w; kx += nkids) {
                sum += input_p[kx] * kernel_p[kx];
            }
            input_p += input_w;
            kernel_p += kernel_w;
        }
        
        // Store result in shared memory
        *(output_p++) = sum;
    }
    __syncthreads();
    
    // Reduce and write back
    if (yy == 0) {
        // Reduce outputs and aggregate results
        for (int k = 1; k < nthreads; k++) {
            for (int i = tid; i < output_w * output_h; i += nthreads) {
                shared_output[i] += shared_output[k * output_h * output_w + i];
            }
        }
        __syncthreads();
        
        // Update existing output and write back
        for (int i = tid; i < output_w * output_h; i += nthreads) {
            output[i] += alpha * shared_output[i];
        }
    }
}