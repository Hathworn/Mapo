#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use loop unrolling for better performance
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < N; i += stride) {
        out[i] = in[i] > 0 ? in[i] : 0;
    }
}