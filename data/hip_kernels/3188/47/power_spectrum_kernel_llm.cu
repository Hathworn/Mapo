#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    float *Ar = A_in + blockIdx.x * ldi;
    float *Aw = A_out + blockIdx.x * ldo;

    int half_length = row_length / 2;

    // Optimize loop to remove unnecessary branching
    for (int idx = thread_id; idx < half_length; idx += blockDim.x * gridDim.x) {
        if (idx != 0) {
            float2 val = reinterpret_cast<float2 *>(Ar)[idx];
            float ret = val.x * val.x + val.y * val.y;
            Aw[idx] = use_power ? ret : sqrtf(ret);
        }
    }

    // Use single thread to handle special case
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}