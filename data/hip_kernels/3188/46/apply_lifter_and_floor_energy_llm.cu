#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void apply_lifter_and_floor_energy( int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    float *feats = features + frame * ldf;

    // Apply lifter coefficients with better parallel efficiency
    if (cepstral_lifter != 0.0f) {
        for (int c = thread_id; c < num_cols; c += blockDim.x) { // Use blockDim.x instead of CU1DBLOCK
            feats[c] *= lifter_coeffs[c]; // Inline lift and multiplication
        }
    }

    // Use thread 0 to apply energy only if needed
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = logf(energy_floor); // Use logf for float

        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        feats[0] = energy;
    }
}