#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Calculate row index
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Calculate column index
    if (i < num_rows && j < num_cols) { // Ensure indices are within bounds
        float f = feats[i * ldf + j]; // Load element
        feats_sq[i * lds + j] = f * f; // Square the element and store
    }
}