#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const float *Ar = A_in + blockIdx.x * ldi;
    float *Aw = A_out + blockIdx.x * ldo;

    int half_length = row_length / 2;

    // Loop through elements using a single warp
    for (int idx = thread_id; idx < half_length; idx += blockDim.x * gridDim.x) {
        // Ignore special case
        if (idx == 0) continue;

        float2 val = reinterpret_cast<const float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret);
    }

    // Handle special case using a single thread in the block
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];
        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}