#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_kernel(const int n, const float alpha, float *y) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop unrolling for better performance
    for (; i < n; i += blockDim.x * gridDim.x * 4) {
        if (i < n) y[i] = alpha;
        if (i + blockDim.x * gridDim.x < n) y[i + blockDim.x * gridDim.x] = alpha;
        if (i + 2 * blockDim.x * gridDim.x < n) y[i + 2 * blockDim.x * gridDim.x] = alpha;
        if (i + 3 * blockDim.x * gridDim.x < n) y[i + 3 * blockDim.x * gridDim.x] = alpha;
    }
}