#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scatter_kernel(int *x_coors, int *y_coors, float *pfe_output, float *scattered_feature, const int max_num_pillars_, const int grid_x_size, const int grid_y_size) {
    // Compute global thread index
    int i_pillar = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread does not exceed the number of pillars
    if (i_pillar < max_num_pillars_) {
        int i_feature = threadIdx.y; // Use second thread dimension for feature

        // Calculate coordinates
        int x_ind = x_coors[i_pillar];
        int y_ind = y_coors[i_pillar];

        // Access and write feature safely
        float feature = pfe_output[i_feature * max_num_pillars_ + i_pillar];
        scattered_feature[i_feature * grid_y_size * grid_x_size + y_ind * grid_x_size + x_ind] = feature;
    }
}