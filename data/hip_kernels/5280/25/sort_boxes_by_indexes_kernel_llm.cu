#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sort_boxes_by_indexes_kernel(float* filtered_box, int* filtered_label, int* filtered_dir, float* box_for_nms, int* indexes, int filter_count, float* sorted_filtered_boxes, int* sorted_filtered_label, int* sorted_filtered_dir, float* sorted_box_for_nms, const int num_box_corners, const int num_output_box_feature) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < filter_count) {
        int sort_index = indexes[tid];

        // Use a loop for copying the features to improve readability and maintainability
        for (int i = 0; i < num_output_box_feature; ++i) {
            sorted_filtered_boxes[tid * num_output_box_feature + i] = filtered_box[sort_index * num_output_box_feature + i];
        }

        sorted_filtered_label[tid] = filtered_label[sort_index];
        sorted_filtered_dir[tid] = filtered_dir[sort_index];

        // Use a loop for copying the box corners
        for (int i = 0; i < num_box_corners; ++i) {
            sorted_box_for_nms[tid * num_box_corners + i] = box_for_nms[sort_index * num_box_corners + i];
        }
    }
}