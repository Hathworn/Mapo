#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(const int n, const float *a, const float *b, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Use loop unrolling to optimize memory access
    for (; i + 3 < n; i += stride * 4) {
        y[i] = a[i] * b[i];
        y[i + 1] = a[i + 1] * b[i + 1];
        y[i + 2] = a[i + 2] * b[i + 2];
        y[i + 3] = a[i + 3] * b[i + 3];
    }
    
    // Process remaining elements
    for (; i < n; i += stride) {
        y[i] = a[i] * b[i];
    }
}