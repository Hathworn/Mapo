#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multi_scale_kernel(const float *data_in, const float *scale, float *data_out, int width, int height) {
    // Calculate single index for thread
    int index = blockDim.x * blockIdx.x + threadIdx.x + (blockDim.y * blockIdx.y + threadIdx.y) * width;
    
    // Boundary check
    if (index < width * height) {
        int y = index / width; // Calculate y-coordinate
        data_out[index] = data_in[index] * scale[y];
    }
}