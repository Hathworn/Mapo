#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void make_pillar_feature_kernel( float* dev_pillar_point_feature_in_coors, float* dev_pillar_point_feature, float* dev_pillar_coors, int* dev_x_coors, int* dev_y_coors, float* dev_num_points_per_pillar, const int max_points, const int num_point_feature, const int grid_x_size) {
    int ith_pillar = blockIdx.x;
    int ith_point = threadIdx.x;

    // Early exit if thread index exceeds points in the pillar
    int num_points_at_this_pillar = dev_num_points_per_pillar[ith_pillar];
    if (ith_point >= num_points_at_this_pillar) return;

    // Precompute reused values
    int x_ind = dev_x_coors[ith_pillar];
    int y_ind = dev_y_coors[ith_pillar];
    int point_offset = ith_point * num_point_feature;
    int pillar_ind = ith_pillar * max_points * num_point_feature + point_offset;
    int coors_ind = (y_ind * grid_x_size + x_ind) * max_points * num_point_feature + point_offset;
    
    // Use unrolled loop for better performance on smaller num_point_feature
    #pragma unroll
    for (int i = 0; i < num_point_feature; ++i) {
        dev_pillar_point_feature[pillar_ind + i] =
            dev_pillar_point_feature_in_coors[coors_ind + i];
    }

    // Directly assign values using constant factor
    dev_pillar_coors[ith_pillar * 4 + 0] = 0.0f;  // batch idx
    dev_pillar_coors[ith_pillar * 4 + 1] = 0.0f;  // z
    dev_pillar_coors[ith_pillar * 4 + 2] = static_cast<float>(y_ind);
    dev_pillar_coors[ith_pillar * 4 + 3] = static_cast<float>(x_ind);
}