#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan_y(int* g_odata, int* g_idata, int n) {
    extern __shared__ int temp[];
    int thid = threadIdx.x;
    int bid = blockIdx.x;
    int gdim = gridDim.x;
    int offset = 1;

    // Improved memory access pattern by calculating indices correctly
    int base_index = bid * n + 2 * thid * gdim;
    temp[2 * thid] = g_idata[base_index];
    temp[2 * thid + 1] = g_idata[base_index + gdim];
    
    for (int d = n >> 1; d > 0; d >>= 1) {
        __syncthreads();
        if (thid < d) {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    
    if (thid == 0) {
        temp[n - 1] = 0;
    }
    
    for (int d = 1; d < n; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (thid < d) {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    // Write results to global memory with optimized index
    g_odata[base_index] = temp[2 * thid + 1];
    g_odata[base_index + gdim] = (2 * thid + 2 == n) ? temp[2 * thid + 1] + g_idata[base_index + gdim] : temp[2 * thid + 2];
}