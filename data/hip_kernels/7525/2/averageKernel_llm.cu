#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16
#define BLOCKSIZE_X 16
#define BLOCKSIZE_Y 16

// STD includes

// CUDA runtime

// Utilities and system includes

// Optimized Kernel Function
__global__ void averageKernel( unsigned char* inputChannel, unsigned char* outputChannel, int imageW, int imageH)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y >= imageH || x >= imageW) return; // Check bounds early

    const unsigned int numElements = ((2 * KERNEL_RADIUS) + 1) * ((2 * KERNEL_RADIUS) + 1);
    unsigned int sum = 0;

    for (int kY = -KERNEL_RADIUS; kY <= KERNEL_RADIUS; kY++) {
        const int curY = y + kY;
        if (curY < 0 || curY >= imageH) continue; // Correct boundary check

        for (int kX = -KERNEL_RADIUS; kX <= KERNEL_RADIUS; kX++) {
            const int curX = x + kX;
            if (curX < 0 || curX >= imageW) continue; // Correct boundary check

            const int curPosition = (curY * imageW + curX);
            sum += inputChannel[curPosition]; // Sum up within boundaries
        }
    }

    outputChannel[y * imageW + x] = (unsigned char)(sum / numElements); // Write result
}