#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16
#define BLOCKSIZE_X 16
#define BLOCKSIZE_Y 16

// Optimize global kernel function to reduce potential bank conflicts and improve performance
__global__ void writeChannelKernel(unsigned char* __restrict__ image, const unsigned char* __restrict__ channel, int imageW, int imageH, int channelToMerge, int numChannels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure within image bounds to avoid out-of-bounds memory access
    if (x < imageW && y < imageH) {
        int posOut = y * (imageW * numChannels) + (x * numChannels) + channelToMerge;
        int posIn = y * imageW + x;
        image[posOut] = channel[posIn];
    }
}