#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16
#define BLOCKSIZE_X 16
#define BLOCKSIZE_Y 16

__global__ void readChannelKernel(unsigned char *image, unsigned char *channel, int imageW, int imageH, int channelToExtract, int numChannels) {
    // Calculate global thread coordinates
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if within image bounds to avoid out-of-bound access
    if (x < imageW && y < imageH) {
        int posIn = y * (imageW * numChannels) + (x * numChannels) + channelToExtract;
        int posOut = y * imageW + x;

        // Extract channel data
        channel[posOut] = image[posIn];
    }
}