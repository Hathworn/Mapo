#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Tri_k(float* a, float* b, float* c, float norm, int n)
{
    int tidx = threadIdx.x % n;
    int Qt = threadIdx.x / n;
    int gb_index_x = Qt + blockIdx.x * (blockDim.x / n);

    float normValue = 1.0f / norm; // Pre-compute to save operations
    float commonValue = ((float)tidx + 1.0f) * normValue;

    b[gb_index_x * n + tidx] = commonValue;
    if (tidx == 0) {
        a[gb_index_x * n + tidx] = 0.0f;
    } else if (tidx == n - 1) {
        a[gb_index_x * n + tidx] = commonValue / 3;
        c[gb_index_x * n + tidx] = 0.0f;
    } else {
        a[gb_index_x * n + tidx] = commonValue / 3;
        c[gb_index_x * n + tidx] = commonValue / 3;
    }
}