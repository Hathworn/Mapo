#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void boundaryCondition_k(float* payoff, size_t spotSize, float strike) {
    size_t state_idx = threadIdx.x; // Get the thread index
    size_t index = state_idx * spotSize; // Compute base index for thread
    payoff[spotSize - 1 + index] = 2 * strike; // Set payoff for max spot
    payoff[index] = 0.0; // Set payoff for min spot
}