#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pcr_k(float* a, float* b, float* c, float* y, int n) {
    // Identifies the thread working within a group
    int tidx = threadIdx.x % n;
    // Identifies the data concerned by the computations
    int Qt = threadIdx.x / n;
    // The global memory access index
    int gb_index_x = Qt + blockIdx.x * (blockDim.x / n);

    // Shared memory
    extern __shared__ float sAds[];

    // Optimized memory calculations
    int nt = 5 * Qt * n;
    float* sa = &sAds[nt];
    float* sb = &sa[n];
    float* sc = &sb[n];
    float* sy = &sc[n];
    int* sl = (int*)&sy[n];

    // Save into shared memory
    sa[tidx] = a[gb_index_x * n + tidx];
    sb[tidx] = b[gb_index_x * n + tidx];
    sc[tidx] = c[gb_index_x * n + tidx];
    sy[tidx] = y[gb_index_x * n + tidx];
    sl[tidx] = tidx;
    __syncthreads();

    // Left/Right indices of the reduction
    int tL = max(tidx - 1, 0); // Use max to simplify boundary condition
    int tR = min(tidx + 1, n - 1); // Use min to simplify boundary condition

    // Local variables
    float aL, bL, cL, yL, aLp, bLp, cLp, yLp;

    for (int i = 0; i < (int)log2f((float)n) + 1; i++) {
        aL = sa[tidx];
        bL = sb[tidx];
        cL = sc[tidx];
        yL = sy[tidx];

        bLp = sb[tL];
        // Reduction phase
        if (fabsf(aL) > EPS) {
            aLp = sa[tL];
            cLp = sc[tL];
            yLp = sy[tL];
            float temp1 = aL / bLp;
            bL -= cLp * temp1;
            yL -= yLp * temp1;
            aL = -aLp * temp1;
        }

        aLp = sa[tR];
        bLp = sb[tR];
        cLp = sc[tR];
        float temp2 = cL / bLp;
        if (fabsf(aLp) > EPS) {
            yLp = sy[tR];
            bL -= aLp * temp2;
            yL -= yLp * temp2;
            cL = -cLp * temp2;
        }
        __syncthreads();
        // Permutation phase
        if (i < (int)log2f((float)n)) {
            sa[d] = aL;
            sb[d] = bL;
            sc[d] = cL;
            sy[d] = yL;
            sl[d] = sl[tidx]; // Simplified permutation
            __syncthreads();
        }
    }

    sy[tidx] = yL / bL;
    __syncthreads();
    y[gb_index_x * n + sl[tidx]] = sy[tidx];
}