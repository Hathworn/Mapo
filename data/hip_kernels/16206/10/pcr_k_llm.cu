#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pcr_k(float a, float b, float c, float* y, int n) {
    // Identifies the thread working within a group
    int tidx = threadIdx.x % n;
    // Identifies the data concerned by the computations
    int Qt = threadIdx.x / n;
    // The global memory access index
    int gb_index_x = Qt + blockIdx.x * (blockDim.x / n);

    // Local integers
    int i, nt, lL, d, tL, tR;
    // Local floats
    float aL, bL, cL, yL, aLp, bLp, cLp, yLp;

    // Shared memory
    extern __shared__ float sAds[];

    nt = 5 * Qt * n;
    d = (n / 2 + (n % 2)) * (tidx % 2) + tidx / 2;
    float* sa = (float*)&sAds[nt];
    float* sb = (float*)&sa[n];
    float* sc = (float*)&sb[n];
    float* sy = (float*)&sc[n];
    int* sl = (int*)&sy[n];

    // Load data into shared memory
    sa[tidx] = a;
    sb[tidx] = b;
    sc[tidx] = c;
    sy[tidx] = y[gb_index_x * n + tidx];
    sl[tidx] = tidx;
    __syncthreads();

    // Left/Right indices of the reduction
    tL = max(tidx - 1, 0); // Ensure non-negative left index
    tR = min(tidx + 1, n - 1); // Ensure the right index doesn't exceed bounds

    for (i = 0; i < (int)log2f((float)n) + 1; i++) {
        lL = sl[tidx];
        aL = sa[tidx];
        bL = sb[tidx];
        cL = sc[tidx];
        yL = sy[tidx];

        bLp = sb[tL];
        // Reduction phase
        if (fabsf(aL) > EPS) {
            aLp = sa[tL];
            cLp = sc[tL];
            yLp = sy[tL];

            bL -= aL * cL / bLp;
            yL -= aL * yLp / bLp;
            aL = -aL * aLp / bLp;
        }

        aLp = sa[tR];
        bLp = sb[tR];
        cLp = sc[tR];
        if (fabsf(aLp) > EPS) {
            yLp = sy[tR];
            bL -= cLp * aLp / bLp;
            yL -= cLp * yLp / bLp;
            cL = -cL * cLp / bLp;
        }
        __syncthreads();
        
        // Permutation phase
        if (i < (int)log2f((float)n)) {
            sa[d] = aL;
            sb[d] = bL;
            sc[d] = cL;
            sy[d] = yL;
            sl[d] = lL;
            __syncthreads();
        }
    }

    sy[tidx] = yL / bL;
    __syncthreads();
    y[gb_index_x * n + sl[tidx]] = sy[tidx];
}