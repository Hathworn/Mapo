#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initPayoff_k(float* payoff, float dx, float Smin, float strike, size_t P1, size_t P2) {
    size_t spot_idx = threadIdx.x;
    size_t state_idx = blockIdx.x;

    // Prefetch dx exponentiation and index computation outside the conditional
    float spot = Smin * expf(spot_idx * dx);
    size_t idx = spot_idx + state_idx * blockDim.x;

    // Use conditional expression without multiplication to simplify
    if (state_idx <= P2 && state_idx >= P1) {
        payoff[idx] = fmaxf(0.0f, spot - strike);
    } else {
        payoff[idx] = 0.0f; // Avoid uninitialized assignments
    }
}