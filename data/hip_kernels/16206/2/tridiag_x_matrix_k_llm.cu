#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tridiag_x_matrix_k(float* p_d, float* p_m, float* p_u, float* u, int n)
{
    // Identifies the thread working within a group
    int tidx = threadIdx.x % n;
    // Identifies the data concerned by the computations
    int Qt = (threadIdx.x - tidx) / n;

    extern __shared__ float sAds[];
    float* su = &sAds[4 * Qt * n];
    float* sp_d = &su[n];
    float* sp_m = &sp_d[n];
    float* sp_u = &sp_m[n];

    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load inputs into shared memory
    su[threadIdx.x] = u[globalIdx];
    sp_d[tidx] = p_d[tidx];
    sp_m[tidx] = p_m[tidx];
    sp_u[tidx] = p_u[tidx];
    __syncthreads();

    float temp;
    // Combine conditions to avoid redundant checks
    if (tidx == 0) {
        temp = sp_m[tidx] * su[tidx] + sp_u[tidx] * su[tidx + 1];
    } else if (tidx == n - 1) {
        temp = sp_d[tidx] * su[tidx - 1] + sp_m[tidx] * su[tidx];
    } else {
        temp = sp_d[tidx] * su[tidx - 1] + sp_m[tidx] * su[tidx] + sp_u[tidx] * su[tidx + 1];
    }

    // Store the result back to global memory
    u[globalIdx] = temp;
}