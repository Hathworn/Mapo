#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void interStep_k(float* payoff, size_t scheduleCounter, float dx, float Smin, size_t P1, size_t P2, float barrier)
{
    size_t spot_idx = blockIdx.x;
    size_t state_idx = threadIdx.x;

    // Avoid shared memory use, calculate directly in registers.
    float temp = 0.0;
    size_t P1_k = P1 > scheduleCounter ? P1 - scheduleCounter : 0;
    float spot = Smin * expf(spot_idx * dx);

    // Reduce conditional complexity by restructuring logic.
    if (state_idx == P2)
    {
        temp = payoff[spot_idx + P2 * gridDim.x] * (spot >= barrier);
    }
    else if (state_idx >= P1_k && state_idx < P2)
    {
        temp = payoff[spot_idx + state_idx * gridDim.x] * (spot >= barrier)
             + payoff[spot_idx + (state_idx + 1) * gridDim.x] * (spot < barrier);
    }
    else if (state_idx == P1_k)
    {
        temp = payoff[spot_idx + P1_k * gridDim.x] * (spot < barrier);
    }

    // Synchronize writes at end only if necessary
    __syncthreads();
    payoff[spot_idx + state_idx * gridDim.x] = temp;
}