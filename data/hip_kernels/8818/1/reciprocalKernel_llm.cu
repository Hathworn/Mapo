#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
    // Use shared memory to optimize memory accesses
    __shared__ float sharedData[1024];

    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < vectorSize) {
        // Load data into shared memory
        sharedData[threadIdx.x] = data[idx];
        __syncthreads();

        // Perform computation
        sharedData[threadIdx.x] = 1.0f / sharedData[threadIdx.x];
        __syncthreads();

        // Store results back to global memory
        data[idx] = sharedData[threadIdx.x];
    }
}