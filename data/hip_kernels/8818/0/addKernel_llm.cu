#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        // Apply loop unrolling for improved instruction throughput
        if (i + 3 < size) {
            c[i] = a[i] + b[i];
            c[i + 1] = a[i + 1] + b[i + 1];
            c[i + 2] = a[i + 2] + b[i + 2];
            c[i + 3] = a[i + 3] + b[i + 3];
        } else {
            c[i] = a[i] + b[i];
        }
    }
}