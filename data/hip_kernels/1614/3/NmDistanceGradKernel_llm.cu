#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NmDistanceGradKernel(int b,int n,const float * xyz1,int m,const float * xyz2,const float * grad_dist1,const int * idx1,float * grad_xyz1,float * grad_xyz2){
    // Calculate global indices for batch and point
    int batch_idx = blockIdx.x + blockIdx.y * gridDim.x;
    int point_idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Bounds check to ensure safe indexing
    if (batch_idx < b && point_idx < n) {
        // Precompute indices for faster access
        int xyz1_idx = (batch_idx * n + point_idx) * 5;
        int j2 = idx1[batch_idx * n + point_idx];
        int xyz2_idx = (batch_idx * m + j2) * 5;

        // Load values into registers for faster access
        float x1 = xyz1[xyz1_idx + 0];
        float y1 = xyz1[xyz1_idx + 1];
        float r1 = xyz1[xyz1_idx + 2];
        float g1 = xyz1[xyz1_idx + 3];
        float b1 = xyz1[xyz1_idx + 4];

        float x2 = xyz2[xyz2_idx + 0];
        float y2 = xyz2[xyz2_idx + 1];
        float r2 = xyz2[xyz2_idx + 2];
        float g2 = xyz2[xyz2_idx + 3];
        float b2 = xyz2[xyz2_idx + 4];

        float g = grad_dist1[batch_idx * n + point_idx] * 2;

        // Perform atomic updates for gradients
        atomicAdd(&grad_xyz1[xyz1_idx + 0], g * (x1 - x2));
        atomicAdd(&grad_xyz1[xyz1_idx + 1], g * (y1 - y2));
        atomicAdd(&grad_xyz1[xyz1_idx + 2], g * (r1 - r2));
        atomicAdd(&grad_xyz1[xyz1_idx + 3], g * (g1 - g2));
        atomicAdd(&grad_xyz1[xyz1_idx + 4], g * (b1 - b2));

        atomicAdd(&grad_xyz2[xyz2_idx + 0], -(g * (x1 - x2)));
        atomicAdd(&grad_xyz2[xyz2_idx + 1], -(g * (y1 - y2)));
        atomicAdd(&grad_xyz2[xyz2_idx + 2], -(g * (r1 - r2)));
        atomicAdd(&grad_xyz2[xyz2_idx + 3], -(g * (g1 - g2)));
        atomicAdd(&grad_xyz2[xyz2_idx + 4], -(g * (b1 - b2)));
    }
}