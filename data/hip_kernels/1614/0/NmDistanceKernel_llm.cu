#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NmDistanceKernel(int b, int n, const float *xyz, int m, const float *xyz2, float *result, int *result_i) {
    const int batch = 512;
    __shared__ float buf[batch * 3];
    
    for (int i = blockIdx.x; i < b; i += gridDim.x) {
        for (int k2 = 0; k2 < m; k2 += batch) {
            int end_k = min(m, k2 + batch) - k2;
            
            for (int j = threadIdx.x; j < end_k * 3; j += blockDim.x) {
                buf[j] = xyz2[(i * m + k2) * 3 + j];
            }
            __syncthreads();
            
            for (int j = threadIdx.x + blockIdx.y * blockDim.x; j < n; j += blockDim.x * gridDim.y) {
                float x1 = xyz[(i * n + j) * 3 + 0];
                float y1 = xyz[(i * n + j) * 3 + 1];
                float z1 = xyz[(i * n + j) * 3 + 2];
                int best_i = 0;
                float best = FLT_MAX; // Initialize with maximum float value

                int end_ka = end_k - (end_k & 3);
                
                for (int k = 0; k < end_ka; k += 4) {
                    #pragma unroll 4 // Unroll loop for better performance
                    for (int x = 0; x < 4; ++x) {
                        float x2 = buf[(k + x) * 3 + 0] - x1;
                        float y2 = buf[(k + x) * 3 + 1] - y1;
                        float z2 = buf[(k + x) * 3 + 2] - z1;
                        float d = x2 * x2 + y2 * y2 + z2 * z2;
                        if (d < best) {
                            best = d;
                            best_i = k + k2 + x;
                        }
                    }
                }

                for (int k = end_ka; k < end_k; k++) {
                    float x2 = buf[k * 3 + 0] - x1;
                    float y2 = buf[k * 3 + 1] - y1;
                    float z2 = buf[k * 3 + 2] - z1;
                    float d = x2 * x2 + y2 * y2 + z2 * z2;
                    if (d < best) {
                        best = d;
                        best_i = k + k2;
                    }
                }

                if (k2 == 0 || result[(i * n + j)] > best) {
                    result[(i * n + j)] = best;
                    result_i[(i * n + j)] = best_i;
                }
            }
            __syncthreads();
        }
    }
}