#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NmDistanceKernel(int b, int n, const float *xyz, int m, const float *xyz2, float *result, int *result_i) {
    const int batch = 512;
    __shared__ float buf[batch * 2];
    for (int i = blockIdx.x; i < b; i += gridDim.x) {
        for (int k2 = 0; k2 < m; k2 += batch) {
            int end_k = min(m, k2 + batch) - k2;
            for (int j = threadIdx.x; j < end_k * 2; j += blockDim.x) {
                buf[j] = xyz2[(i * m + k2) * 2 + j];
            }
            __syncthreads();
            for (int j = threadIdx.x + blockIdx.y * blockDim.x; j < n; j += blockDim.x * gridDim.y) {
                float x1 = xyz[(i * n + j) * 2 + 0];
                float y1 = xyz[(i * n + j) * 2 + 1];
                int best_i = 0;
                float best = FLT_MAX;  // Initialize best distance as maximum possible
                for (int k = 0; k < end_k; k++) {
                    float x2 = buf[k * 2 + 0] - x1;
                    float y2 = buf[k * 2 + 1] - y1;
                    float d = x2 * x2 + y2 * y2;
                    if (d < best) {
                        best = d;
                        best_i = k + k2;
                    }
                }
                if (k2 == 0 || result[i * n + j] > best) {
                    result[i * n + j] = best;
                    result_i[i * n + j] = best_i;
                }
            }
            __syncthreads();
        }
    }
}