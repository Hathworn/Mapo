#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NmDistanceGradKernel(int b, int n, const float *xyz1, int m, const float *xyz2, const float *grad_dist1, const int *idx1, float *grad_xyz1, float *grad_xyz2) {
    // Optimize grid-stride loop for better code efficiency
    int batch_index = blockIdx.x;
    while (batch_index < b) {
        int index = threadIdx.x + blockIdx.y * blockDim.x;
        int stride = blockDim.x * gridDim.y;
        while (index < n) {
            float x1 = xyz1[(batch_index * n + index) * 2 + 0];
            float y1 = xyz1[(batch_index * n + index) * 2 + 1];
            int j2 = idx1[batch_index * n + index];
            float x2 = xyz2[(batch_index * m + j2) * 2 + 0];
            float y2 = xyz2[(batch_index * m + j2) * 2 + 1];
            float g = grad_dist1[batch_index * n + index] * 2;
            atomicAdd(&(grad_xyz1[(batch_index * n + index) * 2 + 0]), g * (x1 - x2));
            atomicAdd(&(grad_xyz1[(batch_index * n + index) * 2 + 1]), g * (y1 - y2));
            atomicAdd(&(grad_xyz2[(batch_index * m + j2) * 2 + 0]), -(g * (x1 - x2)));
            atomicAdd(&(grad_xyz2[(batch_index * m + j2) * 2 + 1]), -(g * (y1 - y2)));
            index += stride;
        }
        batch_index += gridDim.x;
    }
}