#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use blockId.x == 0 and threadId.x == 0 to ensure only one thread updates accuracy
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        *accuracy /= N;
    }
}