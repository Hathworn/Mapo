#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ElementWiseMultiply_CUDA(double *C, double *A, double *B, int rows, int cols)
{
    // Calculate the index using flat index instead of 2D, to simplify boundary check
    int idx = blockDim.x * blockIdx.x + threadIdx.x + (blockDim.y * blockIdx.y + threadIdx.y) * cols;

    // Check boundary with a single condition
    if (idx < rows * cols)
    {
        double a = A[idx];
        double b = B[idx];
        C[idx] = a * b;
    }
}