#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fSigmoid(const float* arguments, float* results, const long size) {
    // Optimize index calculation using blockDim and blockIdx
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds and compute sigmoid
    if (index < size) {
        results[index] = 1.f / (1.f + expf(-arguments[index]));
    }
}