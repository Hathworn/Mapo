#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void fExp(const float* arguments, float* results, const long size) {
    // Optimize index calculation using blockIdx.x and blockIdx.y directly
    const int index = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread-safe memory access and calculate exponent if index is valid
    if(index < size) {
        results[index] = expf(arguments[index]);
    }
}