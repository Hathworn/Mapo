#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"
__global__ void dropoutTrain(const float* arguments, float* dropoutMask, float* results, const float dropoutFraction, const long size) {
    // Calculate global thread index within the grid
    const int index = blockIdx.x + blockIdx.y * gridDim.x + threadIdx.x * gridDim.x * gridDim.y;

    // Ensure the thread operates within the bounds of the data
    if(index < size) {
        // Calculate dropout mask and apply it simultaneously to the results
        dropoutMask[index] = dropoutMask[index] > dropoutFraction;
        results[index] = dropoutMask[index] * arguments[index];
    }
}