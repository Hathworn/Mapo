#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fDerSigmoid(const float* arguments, float* results, const long size) {
    // Calculate global index using 1D grid and block
    const int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (index < size) {
        const float argument = arguments[index];
        results[index] = argument - argument * argument;
    }
}