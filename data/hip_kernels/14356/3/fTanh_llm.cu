#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fTanh(const float* arguments, float* results, const long size) {
    // Calculate global index using block and thread indexing for improved clarity
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds before accessing data
    if (index < size) {
        results[index] = tanh(arguments[index]);
    }
}