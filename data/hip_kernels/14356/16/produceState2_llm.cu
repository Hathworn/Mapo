#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float sigmoid(float x) {
    return 1 / (1 + expf(-x));
}

__global__ void produceState2(const float* arguments, const int argsSize, const float* weights, const int* topology, const int topSize, float* outStates) {
    const int tid = threadIdx.x;
    const int dim = argsSize + topSize;
    extern __shared__ float s[];
    float* states = s;
    bool* ready = (bool*)&states[dim];
    __shared__ int counter;

    int r = tid;
    while (r < dim) {
        ready[r] = false;
        r += blockDim.x;
    }

    if (tid == 0) {
        counter = argsSize;
    }
    if (tid < argsSize) {
        states[tid] = arguments[tid];
        ready[tid] = true;
    }
    __syncthreads();

    while (counter < dim) {
        const int index = counter + tid;
        const int topIndex = index - argsSize;
        if (topIndex < topSize) {
            const int leftBorder = topology[topIndex * 3];
            const int rightBorder = topology[topIndex * 3 + 1];
            const int weightsStart = topology[topIndex * 3 + 2];

            if (rightBorder <= counter) {
                float sum = 0.0f; // Initialize sum
                for (int i = leftBorder; i < rightBorder; ++i) {
                    sum += states[i] * weights[weightsStart + i - leftBorder];
                }
                states[index] = sigmoid(sum);
                ready[index] = true;
            }
        }
        __syncthreads();

        if (tid == 0) {
            for (int i = counter; i < counter + blockDim.x && i < dim; ++i) {
                if (ready[i]) {
                    atomicAdd(&counter, 1); // Use atomic for counter increment
                }
            }
        }
        __syncthreads();
    }

    int n = tid;
    while (n < dim) {
        outStates[n] = states[n];
        n += blockDim.x;
    }
}