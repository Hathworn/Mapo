#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"
__global__ void dropoutTest( const float* arguments, float* results, const float dropoutFraction, const long size ) {
    // Calculate the global thread index more accurately using blockDim
    const int index = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * gridDim.x * blockDim.x);

    // Check if index is within array bounds
    if(index < size) {
        // Apply dropout
        results[index] = arguments[index] * (1.f - dropoutFraction);
    }
}