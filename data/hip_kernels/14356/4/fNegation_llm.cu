#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fNegation(const float* arguments, float* results, const long size) {
    // Calculate the global index for the current thread
    const int index = blockIdx.x + blockIdx.y * gridDim.x + threadIdx.x * gridDim.x * gridDim.y;

    // Ensure the thread operates within the bounds of the data
    if(index < size) {
        results[index] = -arguments[index];
    }
}