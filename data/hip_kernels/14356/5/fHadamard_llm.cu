#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" __global__ void fHadamard(const float* argumentsA, const float* argumentsB, float* results, const long size) {
    // Calculate 1D global index for current thread
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Bounds check to ensure thread does not operate out of allocated memory
    if (index < size) {
        results[index] = argumentsA[index] * argumentsB[index];
    }
}