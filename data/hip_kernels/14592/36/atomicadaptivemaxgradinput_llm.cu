#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void atomicadaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w) {
    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx = threadIdx.x + blockIdx.y * blockDim.x;  // Flatten x and y
    int yy_start = threadIdx.y + blockIdx.z * blockDim.y;  // Flatten y for the starting point
    int yy_step = blockDim.y * gridDim.z; 

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // Compute gradInput
    for(int yy = yy_start; yy < output_h; yy += yy_step) {
        int y_start = (int)floorf(float(yy) / output_h * input_h);

        if (xx < output_w) {  // Ensure we do not access out-of-bounds memory
            int x_start = (int)floorf(float(xx) / output_w * input_w);

            float *ptr_gradInput = gradInput + y_start * input_w + x_start;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float *ptr_ind_x = indices_x + yy * output_w + xx;
            float *ptr_ind_y = indices_y + yy * output_w + xx;
            float z = *ptr_gradOutput;

            int argmax_x = (int)(*ptr_ind_x) - 1;  // Explicit type casting
            int argmax_y = (int)(*ptr_ind_y) - 1;  // Explicit type casting

            // Atomic add since different threads could update same variable
            atomicAdd(&(ptr_gradInput[argmax_x + argmax_y * input_w]), z);
        }
    }
}