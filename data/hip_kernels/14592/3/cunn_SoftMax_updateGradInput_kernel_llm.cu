#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_SoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int nframe, int dim)
{
    __shared__ float buffer[SOFTMAX_THREADS];
    int k = blockIdx.x;
    float *gradInput_k = gradInput + k*dim;
    float *output_k = output + k*dim;
    float *gradOutput_k = gradOutput + k*dim;

    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;

    // Parallel sum
    float local_sum = 0.0f;
    for (int i=i_start; i<i_end; i+=i_step)
        local_sum += gradOutput_k[i] * output_k[i];
    buffer[threadIdx.x] = local_sum;

    __syncthreads();

    // Efficient reduction
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            buffer[threadIdx.x] += buffer[threadIdx.x + s];
        }
        __syncthreads();
    }

    float sum_k = buffer[0];
    for (int i=i_start; i<i_end; i+=i_step)
        gradInput_k[i] = output_k[i] * (gradOutput_k[i] - sum_k);
}