#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_ClassNLLCriterion_updateGradInput_kernel(float *gradInput, float *target, int nframe, int ndim, float grad, int ntarget) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < nframe) { // Ensure thread is within bounds
        for (int j = 0; j < ntarget; ++j) {
            int t = static_cast<int>(target[i * ntarget + j]) - 1;
            if (t >= 0) {
                gradInput[i * ndim + t] = grad; // Assign grad to correct position
            }
        }
    }
}