#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinput(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // compute output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // get thread/block indexes
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    // grid stride for x and y
    int xx_stride = blockDim.x * gridDim.x;
    int yy_stride = blockDim.y * gridDim.y;

    // offset pointers for gradOutput and gradInput
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;

    // fetch weight for this kernel call
    float the_weight = weight[k];

    // row major traversal using grid-stride loop
    for (int yy = blockIdx.y * blockDim.y + threadIdx.y; yy < output_h; yy += yy_stride) {
        for (int xx = blockIdx.x * blockDim.x + threadIdx.x; xx < output_w; xx += xx_stride) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput * the_weight;
            
            // loop unrolling for spatial data
            for (int ky = 0; ky < kH; ky++) {
                #pragma unroll
                for (int kx = 0; kx < kW; kx++) {
                    ptr_gradInput[kx] += z;
                }
                ptr_gradInput += input_w;
            }
        }
    }
}