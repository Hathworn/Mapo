#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y)
{
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor)
{
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float *gradInput_data, float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3)
{
    // Calculate unique thread index globally
    long ii = threadIdx.x + blockDim.x * blockIdx.x + 
              threadIdx.y * blockDim.x * gridDim.x + 
              blockIdx.y * blockDim.y * blockDim.x * gridDim.x;
    
    // Exit if index is out of bounds
    if (ii >= no_elements) return;
    
    // Use threadIdx for offset instead of loop counters
    int i = threadIdx.x % scale_factor;
    int j = threadIdx.y % scale_factor;
    
    // Compute input index
    int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
    
    // Accumulate results directly without loops
    atomicAdd(&gradInput_data[ii], gradOutput_data[ipidx]);
}