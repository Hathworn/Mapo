#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Output size calculation
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    output += o * output_w * output_h;
    input += i * input_w * input_h;

    // Get the good mask for (k,i) (k out, i in)
    float the_weight = weight[k];

    // Initialize to the bias
    float the_bias = bias[k];

    // For all output pixels...
    for (int yy = yy_start; yy < yy_end; yy += yy_step) {
        for (int xx = xx_start; xx < xx_end; xx += xx_step) {
            // Compute the mean of the input image...
            float sum = 0.0f;
            int input_row_offset = yy * dH * input_w;
            int input_col_offset = xx * dW;
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    sum += input[input_row_offset + ky * input_w + input_col_offset + kx];
                }
            }
            // Update output
            output[yy * output_w + xx] = the_weight * sum + the_bias;
        }
    }
}