#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradweight(float *input, float *gradOutput, float *gradWeight, float *gradBias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW, float scale)
{
    // calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // get the current output plane based on block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    // pointers for current plane
    gradOutput += o * output_w * output_h;
    input += i * input_w * input_h;

    // thread ID for sums array
    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    // allocate shared memory for partial sums
    extern __shared__ float shared_sums[];

    // initialize sums
    float local_sum = 0.0f;

    // calculate partial sums
    for (int yy = threadIdx.y; yy < output_h; yy += blockDim.y) {
        for (int xx = threadIdx.x; xx < output_w; xx += blockDim.x) {
            float *ptr_input = input + yy * dH * input_w + xx * dW;
            float z = gradOutput[yy * output_w + xx];
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    local_sum += z * ptr_input[kx];
                }
                ptr_input += input_w;
            }
        }
    }

    // store in shared memory and sync
    shared_sums[tid] = local_sum;
    __syncthreads();

    // accumulate partial sums to gradWeight
    if (tid == 0) {
        float block_sum = 0.0f;
        for (int i = 0; i < blockDim.x * blockDim.y; i++) {
            block_sum += shared_sums[i];
        }
        atomicAdd(&gradWeight[k], scale * block_sum);
    }
    __syncthreads();

    // compute gradBias using atomic operations
    local_sum = 0.0f;
    for (int i = tid; i < output_w * output_h; i += blockDim.x * blockDim.y) {
        local_sum += gradOutput[i];
    }

    // reduce gradBias using atomic add
    shared_sums[tid] = local_sum;
    __syncthreads();

    if (tid == 0) {
        float block_bias_sum = 0.0f;
        for (int i = 0; i < blockDim.x * blockDim.y; i++) {
            block_bias_sum += shared_sums[i];
        }
        atomicAdd(&gradBias[k], scale * block_bias_sum);
    }
}