#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void min_gradInput(float *input, float *output, float *indices, long nrows, long ncols)
{
    // Calculate unique thread identifier for row processing
    long row_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (row_idx < nrows) {
        // Calculate input index for the given row
        long input_offset = row_idx * ncols;

        // Determine min index for gradient backpropagation
        long min_idx = static_cast<long>(indices[row_idx]) - 1;

        // Propagate the gradient to the corresponding input element
        input[input_offset + min_idx] = output[row_idx];
    }
}