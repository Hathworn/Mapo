#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Compute output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx = threadIdx.x;
    int yy = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Select input/output plane
    float *plane_output = output + o * output_w * output_h;
    float *plane_input = input + i * input_w * input_h;

    // Ensure thread operates within bounds
    if (xx < output_w && yy < output_h) {
        float *ptr_input = plane_input + yy * dH * input_w + xx * dW;
        float *ptr_output = plane_output + yy * output_w + xx;
        
        // Compute the mean of the input image
        float sum = 0;
        #pragma unroll
        for (int ky = 0; ky < kH; ky++) {
            #pragma unroll
            for (int kx = 0; kx < kW; kx++) { 
                sum += ptr_input[kx];
            }
            ptr_input += input_w; // Move to next line in the input
        }
        *ptr_output = sum / float(kW * kH);
    }
}