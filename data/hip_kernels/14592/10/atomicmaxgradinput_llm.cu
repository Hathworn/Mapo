#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void atomicmaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW) {
    // Computing the output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on block and thread ID
    int o = blockIdx.x;
    int idX = threadIdx.x + blockIdx.x * blockDim.x;
    int idY = threadIdx.y + blockIdx.y * blockDim.y;

    // Guard clauses to prevent out-of-bounds access
    if (idX >= output_w || idY >= output_h) return;

    // Select input/output plane
    gradOutput = gradOutput + o * output_w * output_h;
    gradInput = gradInput + o * input_w * input_h;
    indices_x = indices_x + o * output_w * output_h;
    indices_y = indices_y + o * output_w * output_h;

    // Compute pointer positions for gradOutput and indices
    float *ptr_gradOutput = gradOutput + idY * output_w + idX;
    float *ptr_ind_x = indices_x + idY * output_w + idX;
    float *ptr_ind_y = indices_y + idY * output_w + idX;

    // Read values and compute the target gradInput position
    float z = *ptr_gradOutput;
    int argmax_x = (*ptr_ind_x) - 1;
    int argmax_y = (*ptr_ind_y) - 1;
    
    // Guard clause for valid indices
    if (argmax_x >= 0 && argmax_y >= 0 && argmax_x < input_w && argmax_y < input_h) {
        // Atomic add since different threads could update the same variable
        atomicAdd(&(gradInput[argmax_x + argmax_y * input_w]), z);
    }
}