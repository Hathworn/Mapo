#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate global indices for accessing data directly
    int timeIndex = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    if (timeIndex < output_w) {
        int batchOffset = blockIdx.x * input_w * input_n;
        float *gradInput_data = gradInput + batchOffset + timeIndex * input_n * dW;
        float *gradOutput_data = gradOutput + batchOffset + timeIndex * input_n;
        float *indices_data = indices + batchOffset + timeIndex * input_n;

        // Use shared memory for faster access
        extern __shared__ float sharedGradOutput[];
        int feat = threadIdx.x;
        
        if (feat < input_n) {
            sharedGradOutput[feat] = gradOutput_data[feat];
        }
        __syncthreads();

        for (feat = 0; feat < input_n; ++feat) {
            int index = (int)indices_data[feat];
            atomicAdd(&gradInput_data[index * input_n + feat], sharedGradOutput[feat]);
        }
    }
}