#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void max_output(float *input, float *output, float *indices, long nrows, long ncols)
{
    // Compute global thread index
    long o = threadIdx.x + blockDim.x * blockIdx.x;
    if (o >= nrows) return;

    // Input offset computation remains unchanged
    long i = o * ncols;
    
    // Initialize max and argmax with first element details
    float max = input[i];
    long argmax = 0;

    // Loop to find max value and its index more efficiently
    for (long ii = 1; ii < ncols; ii++) {
        float val = input[i + ii];
        if (val > max) {
            max = val;
            argmax = ii;
        }
    }

    // Store computed max and its index
    output[o] = max;
    indices[o] = argmax + 1;
}