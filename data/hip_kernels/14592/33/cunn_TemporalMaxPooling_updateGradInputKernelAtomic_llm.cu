#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Block idx is the batch index, thread idx + block idx y * MAX_THREADS is the time index
    
    int globalIndex = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    if (globalIndex >= output_w) return; // Exit if beyond output bounds

    float *gradInput_data = gradInput + blockIdx.x * input_w * input_n + globalIndex * input_n * dW;
    float *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + globalIndex * input_n;
    float *indices_data = indices + blockIdx.x * output_w * input_n + globalIndex * input_n;

    // Unroll the loop for better performance
    for (int feat = 0; feat < input_n; ++feat) {
        atomicAdd(&gradInput_data[(int)indices_data[feat] * input_n + feat], gradOutput_data[feat]);
    }
}