#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_MSECriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, float norm, int nframe, int dim)
{
    // Calculate the absolute index for each thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread index does not exceed the dimension
    if (index < dim * nframe) {
        // Determine the frame
        int k = index / dim;
        
        // Calculate gradInput using the absolute thread index
        gradInput[index] = norm * (input[index] - target[index]);
    }
}