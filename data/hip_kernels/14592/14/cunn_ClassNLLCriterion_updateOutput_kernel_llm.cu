#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_ClassNLLCriterion_updateOutput_kernel(float *output, float *input, float *target, int nframe, int ndim, int sizeAverage, int ntarget) {
    __shared__ float shInputs[NTHREADS];

    // Load shared memory to avoid frequent global memory access
    shInputs[threadIdx.x] = 0.0f;
    __syncthreads();

    float partialSum = 0.0f;
    for (int i = threadIdx.x; i < nframe; i += NTHREADS) {
        for (int j = 0; j < ntarget; ++j) {
            int t = static_cast<int>(target[i * ntarget + j]) - 1;
            if (t >= 0) {
                partialSum += input[i * ndim + t];
            }
        }
    }
    
    // Reduce across threads in a block
    shInputs[threadIdx.x] = partialSum;
    __syncthreads();

    // Perform block-level reduction
    if (threadIdx.x < 32) {
        for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
            shInputs[threadIdx.x] += __shfl_down_sync(0xffffffff, shInputs[threadIdx.x], offset);
        }
    }

    // Final reduction and write to output by the first thread
    if (threadIdx.x == 0) {
        *output = shInputs[0];
        if (sizeAverage) {
            *output /= nframe;
        }
        *output = -(*output);
    }
}