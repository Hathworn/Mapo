#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void maxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
// output size
int output_w = (input_w - kW) / dW + 1;
int output_h = (input_h - kH) / dH + 1;

// compute offsets based on thread/block ID
int o = blockIdx.x;
int i = o;

int xx_start = threadIdx.x;
int xx_end = output_w;
int xx_step = blockDim.x;

int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
int yy_end = output_h;
int yy_step = blockDim.y*gridDim.y;

// select input/output plane
gradOutput = gradOutput + o * output_w * output_h;
gradInput = gradInput + i * input_w * input_h;
indices_x = indices_x + o * output_w * output_h;
indices_y = indices_y + o * output_w * output_h;

// compute gradInput
for (int yy = yy_start; yy < yy_end; yy += yy_step) {
    for (int xx = xx_start; xx < xx_end; xx += xx_step) {
        int idx_output = yy * output_w + xx;
        int idx_input = (int)indices_y[idx_output] - 1;
        int idx_output_x = (int)indices_x[idx_output] - 1;
        float z = gradOutput[idx_output];
        
        // Direct access optimization
        gradInput[idx_output_x + idx_input * input_w] += z;
    }
}
}