#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void max_gradInput(float *input, float *output, float *indices, long nrows, long ncols)
{
    // Calculate the index for the output
    long o = blockIdx.x * blockDim.x + threadIdx.x;
    if (o < nrows) {
        // Calculate input offset
        long i = o * ncols;
        
        // Calculate index and update input
        long idx = indices[o] - 1;
        input[i + idx] = output[o];
    }
}