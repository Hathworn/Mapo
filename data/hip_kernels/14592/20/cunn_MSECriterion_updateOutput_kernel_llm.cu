#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_MSECriterion_updateOutput_kernel(float* output, float *input, float *target, int nframe, int dim, int sizeAverage)
{
    __shared__ float buffer[MSECRITERION_THREADS];
    int k = blockIdx.x;
    float *input_k = input + k * dim;
    float *target_k = target + k * dim;

    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;

    // Initialize buffer with 0 for each thread
    float sum = 0.0f;
    for (int i = i_start; i < i_end; i += i_step)
    {
        float z = input_k[i] - target_k[i];
        sum += z * z;
    }
    buffer[threadIdx.x] = sum;
    __syncthreads();

    // Reduce using parallel reduction
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        if (threadIdx.x < stride)
        {
            buffer[threadIdx.x] += buffer[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Finalize result
    if (threadIdx.x == 0)
    {
        *output = buffer[0];
        if (sizeAverage)
            *output /= dim;
    }
}