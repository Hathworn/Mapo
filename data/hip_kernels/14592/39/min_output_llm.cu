#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void min_output(float *input, float *output, float *indices, long nrows, long ncols)
{
  // output offset:
  long o = threadIdx.x + blockDim.x * blockIdx.x;
  if (o >= nrows) return;

  // input offset:
  long i = o * ncols;

  // move pointers
  input = input + i;

  // compute min using warp parallel reduction
  float min = input[0];
  long argmin = 0;
  for (long stride = 1; stride < ncols; stride *= 2) {
    float val = (i + stride < ncols) ? input[stride] : min;
    long idx = (i + stride < ncols) ? stride : argmin;
    if (val < min) {
      min = val;
      argmin = idx;
    }
  }

  // store
  output[o] = min;
  indices[o] = argmin + 1;
}