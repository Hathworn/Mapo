#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w)
{
    // Precompute values that remain constant during the loop
    int o = blockIdx.x;
    int xx_end = output_w;
    int yy_end = output_h;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += o * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // Compute gradInput using optimized looping
    for (int yy = blockDim.y * blockIdx.y + threadIdx.y; yy < yy_end; yy += yy_step) {
        
        int y_start = __float2int_rd(float(yy) / output_h * input_h); // faster rounding method

        for (int xx = threadIdx.x; xx < xx_end; xx += blockDim.x) {
            
            int x_start = __float2int_rd(float(xx) / output_w * input_w); // faster rounding method

            float *ptr_gradInput = gradInput + y_start * input_w + x_start;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float *ptr_ind_x = indices_x + yy * output_w + xx;
            float *ptr_ind_y = indices_y + yy * output_w + xx;
            float z = *ptr_gradOutput;

            int argmax_x = __float2int_rd(*ptr_ind_x) - 1; // subtract 1 after rounding
            int argmax_y = __float2int_rd(*ptr_ind_y) - 1; // subtract 1 after rounding

            atomicAdd(ptr_gradInput + argmax_x + argmax_y * input_w, z); // use atomicAdd to prevent race conditions
        }
    }
}