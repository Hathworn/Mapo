#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Census_Kernel(unsigned char * MemSrc, unsigned int * MemDst, int eps, int Width, int Height)
{
    // Calculate global thread position
    int globalX = blockIdx.x * blockDim.x + threadIdx.x;
    int globalY = blockIdx.y * blockDim.y + threadIdx.y;
    int GlobalOffset = globalY * Width + globalX;

    if (globalX >= 2 && globalX < Width - 2 && globalY >= 2 && globalY < Height - 2) {
        unsigned char ValueCenter = MemSrc[GlobalOffset];
        unsigned int Census = 0;

        #pragma unroll
        for (int dy = -2; dy <= 2; dy++) {
            #pragma unroll
            for (int dx = -2; dx <= 2; dx++) {
                if (!(dx == 0 && dy == 0)) {
                    unsigned char Value = MemSrc[(globalY + dy) * Width + (globalX + dx)];
                    float Diff = ValueCenter - Value;

                    // Update census bit
                    Census = (Census << 2) | (abs(Diff) <= eps ? 1 : (Diff > eps ? 2 : 0));
                }
            }
        }
        MemDst[GlobalOffset] = Census;
    } else if (globalX < Width && globalY < Height) {
        MemDst[GlobalOffset] = 0; // Handle border pixels
    }
}