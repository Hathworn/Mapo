#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RgbToGray_Kernel(unsigned char * RGB_Image, unsigned char * Gray_Image, int Width, int Height)
{
    // Calculate global thread index
    int globalX = blockIdx.x * blockDim.x + threadIdx.x;
    int globalY = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure the threads handle valid pixels only
    if (globalX < Width && globalY < Height)
    {
        // Calculate offsets
        int OffsetGray = globalY * Width + globalX;
        int OffsetColor = OffsetGray * 3; // Simplified as OffsetGray*3

        // Convert RGB to Grayscale using weighted sum
        Gray_Image[OffsetGray] = (unsigned char)(0.114f * RGB_Image[OffsetColor] 
                            + 0.587f * RGB_Image[OffsetColor + 1] 
                            + 0.299f * RGB_Image[OffsetColor + 2]);
    }
}