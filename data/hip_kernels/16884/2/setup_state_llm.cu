#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Initialize random state with threadIdx.x to ensure unique sequences across threads
    hiprand_init(seed, threadIdx.x, 0, state);
}