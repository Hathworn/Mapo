#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Calculate global thread index
    int index = blockIdx.x * N + threadIdx.x;
    if (threadIdx.x < N) { // Ensure within bounds
        output[index] = -row_sum[blockIdx.x];
    }
}