#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using more efficient memory access
__global__ void ComputeOffsetOfMatrixAB(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, int32_t K_A_B, int32_t N) {
    int32_t row_value = row_sum[blockIdx.x]; // Load row sum for the current block
    for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
        int32_t col_value = col_sum[i]; // Load column sum for the current index
        output[blockIdx.x * N + i] = K_A_B - row_value - col_value; // Calculate offset
    }
}