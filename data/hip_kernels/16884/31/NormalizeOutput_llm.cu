#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Calculate the global index for the current thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a loop with stride only if necessary and unrolling for better performance
    while (idx < num_elements) {
        // Store values into the output array
        to_normalize[idx * 3] = batch_index;
        to_normalize[idx * 3 + 1] = class_index;
        to_normalize[idx * 3 + 2] = static_cast<int64_t>(original[idx]);
        
        // Increment the index for the next iteration, moving in strides
        idx += blockDim.x * gridDim.x;
    }
}