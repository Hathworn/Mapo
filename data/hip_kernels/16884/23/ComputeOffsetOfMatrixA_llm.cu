#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    // Obtain unique global thread ID
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread operates within matrix size
    if (globalIndex < N) {
        output[blockIdx.x * N + globalIndex] = -col_sum[globalIndex];
    }
}