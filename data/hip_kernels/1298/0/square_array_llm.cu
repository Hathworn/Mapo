#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    // Calculate global index with grid-stride loops for optimization
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < N; idx += blockDim.x * gridDim.x) {
        a[idx] = a[idx] * a[idx];
    }
}