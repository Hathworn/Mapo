#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float refineDepthSubPixel(const float3& depths, const float3& sims) {
    float outDepth = -1.0f;

    // Normalize and pre-compute common expressions
    float simM1 = (sims.x + 1.0f) * 0.5f;
    float sim1 = (sims.y + 1.0f) * 0.5f;
    float simP1 = (sims.z + 1.0f) * 0.5f;

    if ((simM1 > sim1) && (simP1 > sim1)) {
        float dispStep = -(simP1 - simM1) / (2.0f * (simP1 + simM1 - 2.0f * sim1));
        float floatDepthM1 = depths.x;
        float floatDepthP1 = depths.z;

        // Calculate a and b for quadratic interpolation
        float b = 0.5f * (floatDepthP1 + floatDepthM1);
        float a = b - floatDepthM1;

        // Compute refined depth
        outDepth = a * dispStep + b;
    }

    return outDepth;
}

__global__ void refine_computeBestDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float3* isims, int isims_p, float3* idpts, int idpts_p, int width, int height, float simThr) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        float3 depths = idpts[y * idpts_p + x];
        float3 sims = isims[y * isims_p + x];

        // Determine best initial depth and similarity
        float outDepth = depths.x;
        float outSim = sims.x;
        if (sims.y < outSim) {
            outDepth = depths.y;
            outSim = sims.y;
        }
        if (sims.z < outSim) {
            outDepth = depths.z;
            outSim = sims.z;
        }

        // Refine depth using subpixel method
        float refinedDepth = refineDepthSubPixel(depths, sims);
        if (refinedDepth > 0.0f) {
            outDepth = refinedDepth;
        }

        // Update output maps based on similarity threshold
        if (outSim < simThr) {
            osim[y * osim_p + x] = outSim;
            odpt[y * odpt_p + x] = outDepth;
        } else {
            osim[y * osim_p + x] = 1.0f;
            odpt[y * odpt_p + x] = -1.0f;
        }
    }
}