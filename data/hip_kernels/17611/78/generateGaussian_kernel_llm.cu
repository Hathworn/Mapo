#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generateGaussian_kernel(float* og, float delta, int radius)
{
    // Calculate index for current thread
    int idx = threadIdx.x;

    // Compute x using the known relationship between thread and block dimensions
    int x = idx - radius;

    // Use precomputed constant to save computations
    float delta2_inv = 1.0f / (2 * delta * delta);
    
    // Optimize exponential calculation
    og[idx] = __expf(-(x * x) * delta2_inv);
}