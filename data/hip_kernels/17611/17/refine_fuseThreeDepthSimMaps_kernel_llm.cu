#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refine_fuseThreeDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float* isimLst, int isimLst_p, float* idptLst, int idptLst_p, float* isimAct, int isimAct_p, float* idptAct, int idptAct_p, int width, int height, float simThr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds for the current thread
    if (x >= width || y >= height) return;

    float dpts[3] = { odpt[y * odpt_p + x], idptLst[y * idptLst_p + x], idptAct[y * idptAct_p + x] };
    float sims[3] = { osim[y * odpt_p + x], isimLst[y * isimLst_p + x], isimAct[y * isimAct_p + x] };

    float outDpt = dpts[0];
    float outSim = simThr;

    // Simplified with ternary operations
    for (int i = 1; i < 3; ++i) {
        if (sims[i] < outSim) {
            outSim = sims[i];
            outDpt = dpts[i];
        }
    }

    osim[y * osim_p + x] = outSim;
    odpt[y * odpt_p + x] = outDpt;
}