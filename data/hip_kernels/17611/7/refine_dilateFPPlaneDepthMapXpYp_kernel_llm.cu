#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refine_dilateFPPlaneDepthMapXpYp_kernel(float* fpPlaneDepthMap, int fpPlaneDepthMap_p, float* maskMap, int maskMap_p, int width, int height, int xp, int yp, float fpPlaneDepth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure valid indices using early exit strategy
    if((x >= width) || (y >= height) || (x + xp < 0) || (y + yp < 0) || (x + xp >= width) || (y + yp >= height))
    {
        return;
    }

    // Simplified variable declaration and access 
    float depth = maskMap[y * maskMap_p + x];
    if(depth > 0.0f)
    {
        fpPlaneDepthMap[(y + yp) * fpPlaneDepthMap_p + (x + xp)] = fpPlaneDepth;
    }
}