#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure i is within bounds
    if (i < filters) {
        float var = variance[i] + 0.00001f;
        float factor = -0.5f * powf(var, -1.5f);
        for (int j = 0; j < batch; ++j) {
            for (int k = 0; k < spatial; ++k) {
                int index = j * filters * spatial + i * spatial + k;
                variance_delta[i] += delta[index] * (x[index] - mean[i]);
            }
        }
        variance_delta[i] *= factor;
    }
}