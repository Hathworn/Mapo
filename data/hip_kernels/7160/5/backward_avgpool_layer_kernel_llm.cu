#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    int b = id / c;

    int out_index = k + c * b;
    int in_offset = h * w * (k + b * c);
    float scaled_out_delta = out_delta[out_index] / (w * h);

    // Use loop unrolling for efficiency
    int i = 0;
    for(; i <= w * h - 4; i += 4) {
        in_delta[in_offset + i] += scaled_out_delta;
        in_delta[in_offset + i + 1] += scaled_out_delta;
        in_delta[in_offset + i + 2] += scaled_out_delta;
        in_delta[in_offset + i + 3] += scaled_out_delta;
    }
    for(; i < w * h; ++i) {
        in_delta[in_offset + i] += scaled_out_delta;
    }
}