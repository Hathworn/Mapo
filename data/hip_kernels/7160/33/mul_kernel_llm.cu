#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Optimize the computation of the global index 'i'
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Add a simple improvement to prevent unnecessary global memory access
    int gridSize = blockDim.x * gridDim.x;
    
    // Use a loop to allow threads to process multiple elements if N is large
    while (i < N) {
        Y[i * INCY] *= X[i * INCX];
        i += gridSize;
    }
}