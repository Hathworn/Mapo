#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (i >= N) return;
    
    // Compute spatial, channel, and batch indices
    int in_s = i % spatial;
    int in_c = (i / spatial) % layers;
    int b = i / (layers * spatial);
    
    // Calculate flattened indices for input and output
    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;
    
    // Perform the data copy based on forward flag
    if (forward) out[i2] = x[i1];
    else out[i1] = x[i2];
}