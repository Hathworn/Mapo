#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use a loop to process elements in a strided manner
    for (int i = idx; i < n; i += stride) {
        if (mask[i] == mask_num) {
            x[i] *= scale;
        }
    }
}