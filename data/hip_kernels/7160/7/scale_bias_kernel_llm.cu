#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Check if offset is in bounds before accessing memory
    if(offset < size) {
        // Calculate index outside of loop to improve memory access efficiency
        int index = (batch * n + filter) * size + offset;
        output[index] *= biases[filter];
    }
}