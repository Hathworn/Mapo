#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified calculation of 'i' for 1D grid/block.
    if(i < n) {
        float t = truth[i];
        float p = pred[i];
        float log_p = (t && p > 0) ? -log(p) : 0; // Prevents log(0) which is undefined.
        error[i] = log_p;
        delta[i] = t - p;
    }
}