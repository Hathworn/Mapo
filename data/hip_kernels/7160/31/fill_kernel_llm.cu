#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimized to use only blockIdx.x for linear block indexing

    // Check if within bounds
    if (i * INCX < N) { // Adjust bounds checking to account for INCX in index calculation
        X[i * INCX] = ALPHA;
    }
}