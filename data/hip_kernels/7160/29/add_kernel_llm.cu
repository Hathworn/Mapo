#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a more efficient loop to process multiple elements using thread
    for(; i < N; i += blockDim.x * gridDim.x)
    {
        X[i * INCX] += ALPHA;
    }
}