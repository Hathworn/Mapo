#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index < N) {  // Simplified boundary check
        int out_w = index % (w * stride);
        int out_h = (index / (w * stride)) % (h * stride);
        int out_c = (index / (w * stride) / (h * stride)) % c;
        int b = index / (w * stride) / (h * stride) / c;

        int in_w = out_w / stride;
        int in_h = out_h / stride;
        int in_index = b * w * h * c + out_c * w * h + in_h * w + in_w;  // Combined redundant calculations

        if (forward) {
            out[index] += scale * x[in_index];  // Data access based on index
        } else {
            atomicAdd(&x[in_index], scale * out[index]);  // Atomic add, reduced pointer arithmetic
        }
    }
}