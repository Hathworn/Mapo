#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate the unique index across grid and block dimensions
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Only perform computations if i is within relevant bounds
    if(i < (NX + NY) * B) {
        int b = i / (NX + NY); // Batch computation by dividing i by (NX + NY)
        int j = i % (NX + NY); // Determine if we're within X or Y portion

        // Ternary operator for assignment based on index position j
        OUT[i] = (j < NX) ? X[b * NX + j] : Y[b * NY + j - NX];
    }
}