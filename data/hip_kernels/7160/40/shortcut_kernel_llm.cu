#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    
    // Unroll division/modulus operations for better performance
    int b = id / (minw * minh * minc);
    int rem = id % (minw * minh * minc);
    int k = rem / (minw * minh);
    rem = rem % (minw * minh);
    int j = rem / minw;
    int i = rem % minw;

    int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
    int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));
    
    // Combine operations to minimize memory access
    float add_val = add[add_index];
    out[out_index] = s1 * out[out_index] + s2 * add_val;
}