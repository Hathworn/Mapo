#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we only process valid indices
    if (i < (NX + NY) * B) {
        int b = i / (NX + NY);  // Batch index
        int j = i % (NX + NY);  // Element index
        
        // Update X or Y based on the element index
        if (j < NX) {
            if (X) atomicAdd(&X[b * NX + j], OUT[i]); // Use atomic to prevent race conditions
        } else {
            if (Y) atomicAdd(&Y[b * NY + j - NX], OUT[i]); // Use atomic to prevent race conditions
        }
    }
}