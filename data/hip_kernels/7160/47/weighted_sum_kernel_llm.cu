#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Optimize index calculation for better performance
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Guard with stride to cover full range of n
    int stride = gridDim.x * blockDim.x;
    for (; i < n; i += stride) {
        c[i] = s[i] * a[i] + (1 - s[i]) * (b ? b[i] : 0);
    }
}