#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    // Shared memory for intermediate calculations
    extern __shared__ float shared_data[];

    int tid = threadIdx.x;
    float* sdata = shared_data + tid * n;

    // Load data into shared memory
    float sum = 0;
    float largest = -INFINITY;
    for (int i = 0; i < n; ++i) {
        sdata[i] = input[i * stride];
        largest = (sdata[i] > largest) ? sdata[i] : largest;
    }
    __syncthreads();

    // Calculate exp and sum
    for (int i = 0; i < n; ++i) {
        float e = expf(sdata[i] / temp - largest / temp);
        sum += e;
        output[i * stride] = e;
    }
    __syncthreads();

    // Normalize the results
    for (int i = 0; i < n; ++i) {
        output[i * stride] /= sum;
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= batch * groups) return;
    int b = id / groups;
    int g = id % groups;

    // Launch device function with allocated shared memory
    softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}