#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (index >= N) return;

    // Pre-compute powers outside of kernel to improve performance if possible
    float one_minus_B1_pow_t = 1.f - powf(B1, t);
    float one_minus_B2_pow_t = 1.f - powf(B2, t);

    float mhat = m[index] / one_minus_B1_pow_t; // Optimize division by constant
    float vhat = v[index] / one_minus_B2_pow_t; // Optimize division by constant

    x[index] += rate * mhat / (sqrtf(vhat) + eps); // Use compound assignment for clarity
}