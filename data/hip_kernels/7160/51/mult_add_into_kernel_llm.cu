#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Use a 2D thread index for better parallel execution
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Unroll the loop to improve memory throughput
    for (int idx = i; idx < n; idx += stride) {
        c[idx] += a[idx] * b[idx];
    }
}