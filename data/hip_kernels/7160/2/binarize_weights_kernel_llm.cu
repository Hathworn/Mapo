#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = gridDim.x * blockDim.x;

    // Process the weights using grid-stride loop
    for (; f < n; f += gridStride) {
        float mean = 0.0f;

        // Unroll the loop for mean calculation
        for (int i = 0; i < size; ++i) {
            mean += fabsf(weights[f * size + i]);
        }
        mean /= size;

        // Unroll the loop for binary assignment
        for (int i = 0; i < size; ++i) {
            binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
        }
    }
}