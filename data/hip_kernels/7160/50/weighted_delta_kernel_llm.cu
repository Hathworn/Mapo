#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Ensure thread index is within bounds
    if (i < n) {
        // Perform operations with appropriate checks
        float dc_val = dc[i];
        float s_val = s[i];

        if (da) {
            da[i] += dc_val * s_val;
        }
        if (db) {
            db[i] += dc_val * (1 - s_val);
        }
        
        ds[i] += dc_val * (a[i] - b[i]);
    }
}