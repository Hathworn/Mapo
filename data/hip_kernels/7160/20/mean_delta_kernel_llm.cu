#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index for filter
    if (i >= filters) return;

    float sum = 0.0f; // Use a local variable for accumulation
    int stride = i * spatial; // Precompute stride for current filter

    for (int j = 0; j < batch; ++j) {
        int batchOffset = j * filters * spatial; // Precompute batch offset
        for (int k = 0; k < spatial; ++k) {
            int index = batchOffset + stride + k;
            sum += delta[index]; // Accumulate deltas
        }
    }

    mean_delta[i] = sum * (-1.f / sqrtf(variance[i] + .00001f)); // Final computation
}