#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (i < n) {
        float p = pred[i];
        float t = truth[i];
        float log_p = logf(p + 1e-7f);             // Use logf for single-precision
        float log_one_minus_p = logf(1.0f - p + 1e-7f); // Use logf for single-precision
        error[i] = -t * log_p - (1.0f - t) * log_one_minus_p;
        delta[i] = t - p;
    }
}