#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;
    
    // Use shared memory for reduction
    __shared__ float shared_sum;
    __shared__ float shared_norm1;
    __shared__ float shared_norm2;
    shared_sum = 0;
    shared_norm1 = 0;
    shared_norm2 = 0;
    __syncthreads();

    int b, i;
    for(b = 0; b < batch; ++b){
        for(i = 0; i < size; ++i){
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            atomicAdd(&shared_sum, output[i1] * output[i2]);
            atomicAdd(&shared_norm1, output[i1] * output[i1]);
            atomicAdd(&shared_norm2, output[i2] * output[i2]);
        }
    }
    __syncthreads();

    norm1 = sqrt(shared_norm1);
    norm2 = sqrt(shared_norm2);
    float norm = norm1 * norm2;
    sum = shared_sum / norm;

    for(b = 0; b < batch; ++b){
        for(i = 0; i < size; ++i){
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            atomicAdd(&delta[i1], - scale * sum * output[i2] / norm);
            atomicAdd(&delta[i2], - scale * sum * output[i1] / norm);
        }
    }
}