#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + 2*pad) / stride;
    int w = (in_w + 2*pad) / stride;
    int c = in_c;
    int area = (size - 1) / stride;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int temp_id = id;
    int j = temp_id % in_w;
    temp_id /= in_w;
    int i = temp_id % in_h;
    temp_id /= in_h;
    int k = temp_id % in_c;
    temp_id /= in_c;
    int b = temp_id;

    int w_offset = -pad;
    int h_offset = -pad;

    float d = 0;
    for(int l = -area; l < area + 1; ++l) {
        for(int m = -area; m < area + 1; ++m) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;
            int out_index = out_w + w * (out_h + h * (k + c * b));
            bool valid = (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h);
            d += (valid && indexes[out_index] == id) ? delta[out_index] : 0; // Accumulate delta for valid and matching index
        }
    }
    atomicAdd(&prev_delta[id], d); // Atomically add delta to prevent write conflicts
}