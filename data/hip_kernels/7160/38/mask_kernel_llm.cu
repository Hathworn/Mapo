#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel for better performance
__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (i < n && mask[i] == mask_num)
    {
        x[i] = val;
    }
}