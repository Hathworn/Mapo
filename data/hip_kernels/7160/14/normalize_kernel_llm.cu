#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Flatten blockIdx and threadIdx for a 1D grid
    int stride = blockDim.x * gridDim.x; // Calculate stride for grid
    for (; index < N; index += stride) { // Loop over data with step size equal to stride
        int f = (index / spatial) % filters; // Calculate filter index
        x[index] = (x[index] - mean[f]) / (sqrtf(variance[f] + .00001f)); // Normalize input
    }
}