#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Optimize thread index calculation for 1D grid
    if (i < N) {
        float value = X[i * INCX];
        if ((value * value) < (ALPHA * ALPHA)) X[i * INCX] = 0;  // Cache value to reduce redundant memory access
    }
}