#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    // Initialize sum to 0 outside the loop
    float temp_sum = 0.0f;
    
    // Use stride to improve memory access pattern
    for (int k = i; k < n * groups; k += groups) {
        temp_sum += x[k];
    }
    
    // Store the calculated sum
    sum[i] = temp_sum;
}