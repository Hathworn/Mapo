#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation
    if (index >= N) return;

    int f = (index / spatial) % filters;

    // Precomputed values for efficiency
    float variance_f = variance[f] + .00001f;
    float inv_sqrt_var = 1.f / sqrtf(variance_f);
    float spatial_batch = spatial * batch;
    float factor = 2.f / spatial_batch;
    
    // Update delta with optimized computation
    delta[index] = delta[index] * inv_sqrt_var +
                   variance_delta[f] * factor * (x[index] - mean[f]) +
                   mean_delta[f] / spatial_batch;
}