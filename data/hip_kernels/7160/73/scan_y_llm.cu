#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan_y(int *g_odata, int *g_idata, int n)
{
    extern __shared__ int temp[];  // allocated on invocation
    int thid = threadIdx.x;
    int bid = blockIdx.x;
    int bdim = blockDim.x;
    int gdim = gridDim.x;

    // Load input into shared memory
    temp[2 * thid] = g_idata[bid + 2 * thid * gdim];
    temp[2 * thid + 1] = g_idata[bid + 2 * thid * gdim + gdim];
    
    int offset = 1;
    
    // Build sum in place up the tree
    for (int d = n >> 1; d > 0; d >>= 1)
    {
        __syncthreads();
        if (thid < d)
        {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    
    // Clear the last element
    if (thid == 0) { temp[n - 1] = 0; }
    
    // Traverse down tree and build scan
    for (int d = 1; d < n; d *= 2)
    {
        offset >>= 1;
        __syncthreads();
        if (thid < d)
        {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    
    __syncthreads();
    
    // Write results to device memory
    g_odata[bid + 2 * thid * gdim] = temp[2 * thid + 1];
    int second_ind = 2 * thid + 2;
    if (second_ind == bdim * 2)
    {
        g_odata[bid + 2 * thid * gdim + gdim] = temp[2 * thid + 1] + g_idata[bid + 2 * thid * gdim + gdim];
    }
    else
    {
        g_odata[bid + 2 * thid * gdim + gdim] = temp[2 * thid + 2];
    }
}