#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Use a 1D block and grid infrastructure to simplify index calculation
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds before applying operation
    if (idx < N) {
        Y[idx * INCY] = pow(X[idx * INCX], ALPHA);
    }
}