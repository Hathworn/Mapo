#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation to work only in x-dimension
    int stride = blockDim.x * gridDim.x; // Use stride for grid-stride loop

    for (int i = idx; i < n; i += stride) { // Utilize grid-stride loop for accessing all elements
        float diff = truth[i] - pred[i];
        error[i] = diff * diff; // Compute squared error
        delta[i] = diff; // Compute delta
    }
}