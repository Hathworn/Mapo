#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Flatten the block grid
    int stride = gridDim.x * blockDim.x;             // Define the stride for the loop

    for (int i = idx; i < N; i += stride) {          // Loop with stride for better workload distribution
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}