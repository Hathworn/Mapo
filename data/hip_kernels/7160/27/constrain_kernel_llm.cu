#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation to 1D grid
    if (i < N) 
    {
        int index = i * INCX; // Precaution for performance: reduce repetitive arithmetic calculation
        X[index] = fminf(ALPHA, fmaxf(-ALPHA, X[index]));
    }
}