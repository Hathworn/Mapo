#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel implementation
__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride-based approach to cover entire range
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x)
    {
        X[idx * INCX] *= ALPHA;
    }
}