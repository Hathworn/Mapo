#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    for(i = 0; i < n; ++i){
        float val = input[i*stride];
        largest = (val > largest) ? val : largest;
    }
    for(i = 0; i < n; ++i){
        float e = expf(input[i*stride]/temp - largest/temp);
        sum += e;
        output[i*stride] = e;
    }
    for(i = 0; i < n; ++i){
        output[i*stride] /= sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;  // Compute total threads

    while (id < spatial * batch * groups) {  // Iterate over necessary elements
        int s = id % spatial;
        int group_idx = id / spatial;
        int g = group_idx % groups;
        int b = group_idx / groups;
        int goff = group_offset[g] * spatial;
        int boff = b * stride;
        
        // Invoke softmax_device for the specific group and batch
        softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
        
        id += total_threads;  // Move to the next set of elements for this thread
    }
}