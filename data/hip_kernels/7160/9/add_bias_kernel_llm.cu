#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    // Calculate the global index using built-in functions
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Total number of threads

    for (int idx = index; idx < batch * n * size; idx += stride) {
        int i = idx % size;
        idx /= size;
        int j = idx % n;
        idx /= n;
        int k = idx;

        // Add bias only using the thread IDs
        output[(k * n + j) * size + i] += biases[j];
    }
}