#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float sum = 0.0f;  // Reduced global memory access
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += x[index];
        }
    }
    mean[i] = sum * scale;  // Compute mean once outside loops
}