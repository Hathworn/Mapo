#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Calculate the unique global thread index for filters
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return; // Early exit if thread index is out of filter range

    float scale = 1.f / (batch * spatial - 1);
    float var_sum = 0.0f;
    
    // Unrolling loops for improved performance on GPU
    for (int j = 0; j < batch; ++j) {
        int base_index = j * filters * spatial + i * spatial;
        for (int k = 0; k < spatial; ++k) {
            int index = base_index + k;
            float diff = x[index] - mean[i];
            var_sum += diff * diff;
        }
    }
    
    variance[i] = var_sum * scale; // Apply scaling factor to the variance sum
}