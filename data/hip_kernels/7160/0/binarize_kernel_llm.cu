#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation for 1D grid.
    if (i < n) // Use < instead of >= to improve readability.
    {
        binary[i] = (x[i] >= 0) ? 1 : -1; // Conditionally assign value based on input.
    }
}