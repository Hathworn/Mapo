#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate global index using optimized 1D grid for potentially improved performance
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure that only relevant threads within bounds perform the memory copy operation
    if (i < N)
    {
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
    }
}