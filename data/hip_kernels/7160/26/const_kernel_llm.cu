#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride to cover entire array
    int stride = gridDim.x * blockDim.x;
    
    // Loop with stride to ensure full coverage
    for (; i < N; i += stride)
    {
        X[i * INCX] = ALPHA;
    }
}