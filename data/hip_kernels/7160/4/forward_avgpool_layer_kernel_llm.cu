#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    // Using shared memory for block-wise accumulation
    extern __shared__ float shared_data[];

    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int k = id % c;
    id /= c;
    int b = id;

    int tid = threadIdx.x;
    shared_data[tid] = 0;

    // Calculate pool region sum in parallel using block of threads
    for (int i = tid; i < w * h; i += blockDim.x) {
        int in_index = i + h * w * (k + b * c);
        shared_data[tid] += input[in_index];
    }

    // Sum reduction within block
    __syncthreads();
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }

    // Write the result for this block to the output
    if (tid == 0) {
        output[k + c * b] = shared_data[0] / (w * h);
    }
}