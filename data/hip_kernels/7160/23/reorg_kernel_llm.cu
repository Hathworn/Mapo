#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Calculate unique global thread index
    int global_index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Check if the thread is within bounds
    if (global_index >= N) return;

    // Compute in_w, in_h, in_c, and batch b
    int temp = global_index;
    int in_w = temp % w;
    temp /= w;
    int in_h = temp % h;
    temp /= h;
    int in_c = temp % c;
    temp /= c;
    int b = temp % batch;

    // Calculate corresponding output channel and offset
    int out_c = c / (stride * stride);
    int offset = in_c / out_c;

    // Efficient w2 and h2 calculation without division
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;

    // Calculate the output index
    int out_index = w2 + w * stride * (h2 + h * stride * (in_c % out_c + out_c * b));

    // Conditional data transfer
    if (forward) {
        out[out_index] = x[global_index];
    } else {
        out[global_index] = x[out_index];
    }
}