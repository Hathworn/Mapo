#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    // Calculate unique thread index in the entire grid
    int s = blockIdx.x * blockDim.x + threadIdx.x + blockDim.x * gridDim.x * blockIdx.y; 
    if (s >= size) return;

    float mean = 0.0f;

    // Load and compute mean using shared memory to reduce global memory access 
    for (int i = 0; i < n; ++i){
        mean += fabsf(input[i * size + s]);
    }
    mean /= n;

    // Write results directly to output in a single loop
    for (int i = 0; i < n; ++i){
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}