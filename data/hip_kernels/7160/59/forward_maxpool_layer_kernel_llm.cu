#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    // Calculate output dimensions
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int c = in_c;

    // Calculate the unique thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Decode id into multi-dimensional indices
    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -pad;
    int h_offset = -pad;

    int out_index = j + w * (i + h * (k + c * b));
    float max = -INFINITY;
    int max_i = -1;

    // Unroll loops if necessary and eliminate redundant computation
    for (int l = 0; l < size; ++l) {
        for (int m = 0; m < size; ++m) {
            int cur_h = h_offset + i * stride + l;
            int cur_w = w_offset + j * stride + m;
            if (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w) {
                int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
                float val = input[index];
                if (val > max) {
                    max = val;
                    max_i = index;
                }
            }
        }
    }
    // Store results in output array
    output[out_index] = max;
    indexes[out_index] = max_i;
}