#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod, const size_t lod_len, size_t* out_lod0) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure the thread is within bounds and eliminate conditional within the loop
    if (index < lod_len) {
        size_t idx = in_lod[index];
        out_lod0[index] = idx >= lod_len ? 0 : in_lod[index] - num_erased[idx];
    }
}