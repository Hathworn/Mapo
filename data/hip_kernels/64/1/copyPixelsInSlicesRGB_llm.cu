#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyPixelsInSlicesRGB(float *ptrinput0, float *ptrkslices0, int dH, int dW, int kH, int kW, int size1, int size2, int isize1, int isize2, int nInputPlane, int padleft, int padright, int padup, int paddown, int inputstr0, int kslicesstr0, int batchsize)
{
    // Calculate pixel coordinates
    const int pixi = blockIdx.x;
    const int pixj = blockIdx.y * blockDim.y + threadIdx.y;
    const int tidx = threadIdx.x;
    const int batchindex = blockIdx.z * blockDim.z + threadIdx.z;

    if (batchindex >= batchsize || pixj >= isize2 + padleft + padright) return;

    // Shared memory for better performance in shared calculations
    __shared__ int shared_data[32 * 5]; // Optimized structure to hold all shared values
    int * _imin = shared_data + 32 * 0;
    int * _jmin = shared_data + 32 * 1;
    int * _imax = shared_data + 32 * 2;
    int * _jmax = shared_data + 32 * 3;
    int * _offsets = shared_data + 32 * 4;

    // Calculate bounds and offsets
    if (threadIdx.z == 0)
    {
        int imin = max((pixi - (kH - 1) + (dH - 1)) / dH, 0);
        int jmin = max((pixj - (kW - 1) + (dW - 1)) / dW, 0);
        int imax = min(pixi / dH, size1 - 1);
        int jmax = min(pixj / dW, size2 - 1);

        if (threadIdx.x == 0 && threadIdx.y == 0)
        {
            _imin[tidx] = imin;
            _imax[tidx] = imax;
        }
        if (threadIdx.x == 0)
        {
            _jmin[threadIdx.y] = jmin;
            _jmax[threadIdx.y] = jmax;
        }

        int inputoffset = inputstr0 * blockIdx.z * blockDim.z + ((pixi - padup) * isize2 + (pixj - padleft)) * nInputPlane;
        int ksliceoffset = kslicesstr0 * blockIdx.z * blockDim.z + ((imin * size2 + jmin) * kH * kW + (pixi - imin * dH) * kW + (pixj - jmin * dW)) * nInputPlane;

        _offsets[threadIdx.y * 3] = inputoffset;
        _offsets[threadIdx.y * 3 + 1] = ksliceoffset;
    }

    __syncthreads();

    // Extract values from shared memory
    int imin = _imin[tidx];
    int jmin = _jmin[threadIdx.y];
    int imax = _imax[tidx];
    int jmax = _jmax[threadIdx.y];
    int inputoffset = _offsets[threadIdx.y * 3];
    int ksliceoffset = _offsets[threadIdx.y * 3 + 1];

    // Update pointers
    ptrinput0 += inputoffset + inputstr0 * threadIdx.z;
    ptrkslices0 += ksliceoffset + kslicesstr0 * threadIdx.z;

    // Calculate strides and padding flag
    const int stridej = (kH * kW - dW) * nInputPlane;
    const int stridei = (size2 * kH - dH) * kW * nInputPlane - (jmax - jmin + 1) * stridej;
    bool zeropad = pixi < padup || pixi > isize1 - 1 + padup || pixj < padleft || pixj > isize2 - 1 + padleft;

    // Read pixel value
    float pixvalue = zeropad ? 0 : ptrinput0[tidx];

    // Write to memory with calculated strides
    for (int i = imin; i <= imax; i++) {
        for (int j = jmin; j <= jmax; j++) {
            ptrkslices0[tidx] = pixvalue;
            ptrkslices0 += stridej;
        }
        ptrkslices0 += stridei;
    }
}