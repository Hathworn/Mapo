#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyBiasToOutputs(float *ptrbias, float *ptroutput, const int size1, const int size2, const int nOutputPlane, const int linestride, const int imstride)
{
    // Calculate 3D index for each thread
    const int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    const int tidy = blockIdx.y;
    const int tidz = blockIdx.z;

    // Ensure tidx is within range before proceeding
    if (tidx < nOutputPlane) {
        float val = ptrbias[tidx];
        ptroutput += tidz * imstride + tidy * linestride;

        // Optimize loop to unroll for better performance
        #pragma unroll
        for (int k = 0; k < size2; k++) {
            ptroutput[k * nOutputPlane + tidx] = val;
        }
    }
}