#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

unsigned char *pdata; // pointer to data content

__global__ void processData(unsigned char *Da, int* filter)
{
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int bn = blockDim.x;
    int gid = bx * bn + tx;

    __shared__ int sfilter[3][3];
    __shared__ int sR[3][512];
    __shared__ int sG[3][512];
    __shared__ int sB[3][512];
    __shared__ int sRsum[512];
    __shared__ int sGsum[512];
    __shared__ int sBsum[512];

    // Load filter into shared memory using a single transaction
    if (tx < 9) {
        sfilter[tx / 3][tx % 3] = filter[tx];
    }
    __syncthreads();

    // Direct assignment for edge pixels
    if (bx == 0 || bx == 511 || tx == 0 || tx == bn - 1) {
        sRsum[tx] = Da[gid * 3];
        sGsum[tx] = Da[gid * 3 + 1];
        sBsum[tx] = Da[gid * 3 + 2];
    } else {
        // Load relevant rows into shared memory (except edge cases)
        sR[0][tx] = Da[(gid - bn) * 3];
        sR[1][tx] = Da[gid * 3];
        sR[2][tx] = Da[(gid + bn) * 3];

        sG[0][tx] = Da[(gid - bn) * 3 + 1];
        sG[1][tx] = Da[gid * 3 + 1];
        sG[2][tx] = Da[(gid + bn) * 3 + 1];

        sB[0][tx] = Da[(gid - bn) * 3 + 2];
        sB[1][tx] = Da[gid * 3 + 2];
        sB[2][tx] = Da[(gid + bn) * 3 + 2];
        __syncthreads();

        // Compute convolution for non-edge pixels
        if (tx != 0 && tx != bn - 1) {
            int rSum = 0, gSum = 0, bSum = 0;
            #pragma unroll
            for (int i = 0; i < 3; i++) {
                for (int j = -1; j <= 1; j++) {
                    rSum += sR[i][tx + j] * sfilter[i][j + 1];
                    gSum += sG[i][tx + j] * sfilter[i][j + 1];
                    bSum += sB[i][tx + j] * sfilter[i][j + 1];
                }
            }

            // Normalize and clamp values
            sRsum[tx] = min(max(rSum / filter[9], 0), 255);
            sGsum[tx] = min(max(gSum / filter[9], 0), 255);
            sBsum[tx] = min(max(bSum / filter[9], 0), 255);
        }
    }

    __syncthreads();

    // Combine results back into global memory
    Da[gid * 3] = sRsum[tx];
    Da[gid * 3 + 1] = sGsum[tx];
    Da[gid * 3 + 2] = sBsum[tx];
}