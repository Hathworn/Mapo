#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

#define TILE_WIDTH 2

// main fn
__global__ void MatrixMult(int m, int n, int k, float *a, float *b, float *c)
{
    // Use shared memory for efficient data access
    __shared__ float sharedA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sharedB[TILE_WIDTH][TILE_WIDTH];

    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    float temp = 0.0;

    // Loop over tiles of the input matrices
    for (int tileIdx = 0; tileIdx < (n - 1) / TILE_WIDTH + 1; ++tileIdx) {
        // Load tiles into shared memory
        if (row < m && tileIdx * TILE_WIDTH + threadIdx.x < n) {
            sharedA[threadIdx.y][threadIdx.x] = a[row * n + tileIdx * TILE_WIDTH + threadIdx.x];
        } else {
            sharedA[threadIdx.y][threadIdx.x] = 0.0;
        }
        
        if (col < k && tileIdx * TILE_WIDTH + threadIdx.y < n) {
            sharedB[threadIdx.y][threadIdx.x] = b[(tileIdx * TILE_WIDTH + threadIdx.y) * k + col];
        } else {
            sharedB[threadIdx.y][threadIdx.x] = 0.0;
        }
        
        __syncthreads();

        // Calculate partial sum for this tile
        for (int i = 0; i < TILE_WIDTH; ++i) {
            temp += sharedA[threadIdx.y][i] * sharedB[i][threadIdx.x];
        }
        
        __syncthreads();
    }

    // Write result to global memory
    if (row < m && col < k) {
        c[row * k + col] = temp;
    }
}