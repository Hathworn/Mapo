#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecSum(double* devIn, int pow_step, int n)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure that the index is within bounds and the second element is valid
    if (i < n / pow_step)
    {
        // Perform the addition using a single memory read/write per element
        int index = pow_step * i;
        devIn[index] += devIn[index + (pow_step / 2)];
    }
}