#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_multiply(float *a, float *b, float *c, int num, size_t width)
{
    // create shorthand names for threadIdx & blockIdx
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;

    // allocate 2D tiles in __shared__ memory
    __shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
    __shared__ float s_b[TILE_WIDTH][TILE_WIDTH];

    // calculate the row & column index of the element
    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    float result = 0;

    // loop over the tiles of the input in phases
    for(int i = 0; i < (width + TILE_WIDTH - 1) / TILE_WIDTH; ++i)
    {
        // collaboratively load tiles into __shared__
        if (row < width && i*TILE_WIDTH + tx < width)
            s_a[ty][tx] = a[row*width + i*TILE_WIDTH + tx];
        else
            s_a[ty][tx] = 0.0;

        if (col < width && i*TILE_WIDTH + ty < width)
            s_b[ty][tx] = b[(i*TILE_WIDTH + ty)*width + col];
        else
            s_b[ty][tx] = 0.0;

        // sync threads within block
        __syncthreads();

        // do dot product between row of s_a and column of s_b
        #pragma unroll  // unroll loop to increase performance
        for(int k = 0; k < TILE_WIDTH; ++k)
        {
            result += s_a[ty][k] * s_b[k][tx];
        }

        // sync threads within block
        __syncthreads();
    }

    if (row < width && col < width)
    {
        c[row*num + col] = result;
    }
}