#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void imgBlur(float* imgIn, float* imgOut, int imageWidth, int imageHeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < imageWidth && idy < imageHeight)
    {
        float sum = imgIn[idy * imageWidth + idx]; // Correct index calculation

        // Precompute boundaries to avoid repeated checks
        int left = max(0, idx-1);
        int right = min(imageWidth-1, idx+1);
        int up = max(0, idy-1);
        int down = min(imageHeight-1, idy+1);

        // Simplifying neighbor access
        sum += imgIn[up * imageWidth + left];
        sum += imgIn[up * imageWidth + idx];
        sum += imgIn[up * imageWidth + right];
        sum += imgIn[idy * imageWidth + left];
        sum += imgIn[idy * imageWidth + right];
        sum += imgIn[down * imageWidth + left];
        sum += imgIn[down * imageWidth + idx];
        sum += imgIn[down * imageWidth + right];

        imgOut[idy * imageWidth + idx] = sum / 9.0f; // Simplified divisor
    }
}