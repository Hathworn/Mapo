#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMul(int *A, int *B, int *C, int n)
{
    // Calculate global row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int num = n;
    if (row < num && col < num)
    {
        // Utilize register to accumulate result
        int Cvalue = 0;
        for (int i = 0; i < num; i++)
        {
            // Coalesce memory access for A and B
            Cvalue += A[row * num + i] * B[i * num + col];
        }
        // Directly store result to global memory
        C[row * num + col] = Cvalue;
    }
}