#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(const float *A, const float *B, float *C, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Efficient check to ensure thread is within bounds
    if (i >= n) return;

    C[i] = A[i] + B[i];
}