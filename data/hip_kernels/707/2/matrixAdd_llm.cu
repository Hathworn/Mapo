#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixAdd(float *A, float *B, float *C, int n)
{
    // Calculate the row and column index for the element
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check bounds and perform addition
    if (row < n && col < n)
    {
        int i = row * n + col;
        C[i] = A[i] + B[i];
    }
}