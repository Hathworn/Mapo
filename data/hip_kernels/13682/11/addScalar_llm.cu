#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addScalar(int a, int b, int* ptrC)
{
    // Only execute the addition and print if it's the first thread
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        *ptrC = a + b;
        printf("[GPU] %d + %d = %d", a, b, *ptrC);
    }
}