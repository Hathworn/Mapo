#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation for 1D grid
    if (i >= filters) return;

    float temp_variance_delta = 0.0f; // Local temporary variable to reduce global memory accesses
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            temp_variance_delta += delta[index] * (x[index] - mean[i]);
        }
    }
    
    // Avoid repeated pow calculation
    float inv_variance_pow = -.5f * rsqrtf(variance[i] + .000001f);
    variance_delta[i] = temp_variance_delta * inv_variance_pow;
}