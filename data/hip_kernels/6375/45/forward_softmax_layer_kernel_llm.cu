#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_softmax_layer_kernel(int n, int batch, float *input, float temp, float *output)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if(b >= batch) return;

    extern __shared__ float buffer[]; // Shared memory for reduction
    float *shared_input = buffer;
    float *shared_exps = buffer + n;

    float sum = 0;
    float largest = -INFINITY;

    // Load and find the largest value
    for(int i = threadIdx.x; i < n; i += blockDim.x){
        shared_input[i] = input[i + b * n];
        atomicMax(&largest, shared_input[i]);
    }
    __syncthreads(); // Ensure all threads have loaded data

    // Compute exponentials and sum them up
    for(int i = threadIdx.x; i < n; i += blockDim.x){
        shared_exps[i] = expf((shared_input[i] - largest) / temp);
        atomicAdd(&sum, shared_exps[i]);
    }
    __syncthreads();

    sum = __syncthreads_or(sum); // Broadcast sum to all threads

    // Calculate final output
    for(int i = threadIdx.x; i < n; i += blockDim.x){
        output[i + b * n] = shared_exps[i] / sum;
    }
}