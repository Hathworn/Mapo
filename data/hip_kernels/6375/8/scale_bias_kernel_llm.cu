#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int batch = blockIdx.z;
    int filter = blockIdx.y;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Combined offset computation
    int offset = (batch * n + filter) * size + idx;
    
    if (idx < size)
        output[offset] *= biases[filter];
}