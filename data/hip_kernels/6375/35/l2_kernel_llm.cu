#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float diff = truth[i] - pred[i];
        error[i] = diff * diff; // Compute squared error
        delta[i] = diff; // Compute error delta
    }
}