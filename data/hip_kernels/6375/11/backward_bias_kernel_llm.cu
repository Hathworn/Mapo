#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Use grid-stride loop for better device utilization
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index];
        }
    }

    part[p] = sum;
    __syncthreads();

    // Reduce within block using a single thread
    if (p == 0) {
        sum = 0;
        for (int i = 0; i < BLOCK; ++i) {
            sum += part[i];
        }
        atomicAdd(&bias_updates[filter], sum); // Use atomicAdd for thread safety
    }
}