#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_filters_kernel(float *filters, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Reduce accesses by caching in shared memory
    __shared__ float shared_filters[1024];

    // Load data into shared memory
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_filters[i] = abs(filters[f * size + i]);
    }
    __syncthreads();

    // Calculate mean with reduction
    float mean = 0;
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        mean += shared_filters[i];
    }
    mean /= size;
    __syncthreads();

    // Binarize filters
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (filters[f * size + i] > 0) ? mean : -mean;
    }
}