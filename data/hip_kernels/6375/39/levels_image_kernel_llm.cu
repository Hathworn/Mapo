#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    float h = hsv.x;
    float s = hsv.y;
    float v = hsv.z;

    float r, g, b;
    float f, p, q, t;

    if (s == 0) {
        r = g = b = v;
    } else {
        int index = (int) floorf(h);
        f = h - index;
        p = v * (1 - s);
        q = v * (1 - s * f);
        t = v * (1 - s * (1 - f));

        switch (index) { // Simplify using switch-case
            case 0: r = v; g = t; b = p; break;
            case 1: r = q; g = v; b = p; break;
            case 2: r = p; g = v; b = t; break;
            case 3: r = p; g = q; b = v; break;
            case 4: r = t; g = p; b = v; break;
            default: r = v; g = p; b = q; break;
        }
    }

    r = fminf(fmaxf(r, 0.0f), 1.0f);
    g = fminf(fmaxf(g, 0.0f), 1.0f);
    b = fminf(fmaxf(b, 0.0f), 1.0f);

    return make_float3(r, g, b);
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    float r = rgb.x;
    float g = rgb.y;
    float b = rgb.z;

    float h, s, v;
    float max = fmaxf(fmaxf(r, g), b); // Use fmaxf for max
    float min = fminf(fminf(r, g), b); // Use fminf for min
    float delta = max - min;
    v = max;

    if (max == 0) {
        s = 0;
        h = -1;
    } else {
        s = delta / max;
        if (r == max) {
            h = (g - b) / delta;
        } else if (g == max) {
            h = 2 + (b - r) / delta;
        } else {
            h = 4 + (r - g) / delta;
        }
        if (h < 0) h += 6;
    }

    return make_float3(h, s, v);
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int size = batch * w * h;
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;

    int x = id % w;
    int y = (id / w) % h;
    id /= h; // Simplify id calculation

    float rshift = rand[0];
    float gshift = rand[1];
    float bshift = rand[2];
    float r0 = rand[8 * id + 0];
    float r1 = rand[8 * id + 1];
    float r2 = rand[8 * id + 2];
    float r3 = rand[8 * id + 3];

    saturation = r0 * (saturation - 1) + 1;
    if (r1 > .5) saturation = 1. / saturation; // Combine condition immediately

    exposure = r2 * (exposure - 1) + 1;
    if (r3 > .5) exposure = 1. / exposure; // Combine condition immediately

    size_t offset = id * h * w * 3;
    image += offset;
    float r = image[x + w * (y + h * 0)];
    float g = image[x + w * (y + h * 1)];
    float b = image[x + w * (y + h * 2)];
    float3 rgb = make_float3(r, g, b);

    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }
    
    // Consolidate image updates
    image[x + w * (y + h * 0)] = rgb.x * scale + translate + (rshift - .5) * shift;
    image[x + w * (y + h * 1)] = rgb.y * scale + translate + (gshift - .5) * shift;
    image[x + w * (y + h * 2)] = rgb.z * scale + translate + (bshift - .5) * shift;
}