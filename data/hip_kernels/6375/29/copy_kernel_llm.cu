#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY) {
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Stride loop to allow more threads to participate
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        Y[idx * INCY + OFFY] = X[idx * INCX + OFFX];
    }
}