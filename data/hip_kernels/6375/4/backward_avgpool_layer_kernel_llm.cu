#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if(id >= n * c * w * h) return; // Expanded the range check to include all indexes

    int in_index = id;
    int k = id / (w * h) % c; // Calculate k from the id
    int b = id / (w * h * c); // Calculate b from the id

    int out_index = k + c * b; // Output index calculation
    in_delta[in_index] += out_delta[out_index] / (w * h); // Compute and accumulate delta
}