#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        // Use a pointer to avoid recalculating index offsets
        float *x_ptr = x + j * spatial * filters + filter * spatial;
        for(i = id; i < spatial; i += threads){
            local[id] += x_ptr[i];
        }
    }

    // Use atomic addition for thread-safe mean calculation
    atomicAdd(&mean[filter], local[id] / (spatial * batch));
}