#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use block and thread indexing to calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Check the condition within the loop for slightly better performance
    if(i < N) 
    {
        // Unroll the loop where possible for better memory access
        #pragma unroll
        for(int offset = 0; offset < INCX; ++offset)
        {
            X[i * INCX + offset] = ALPHA;
        }
    }
}