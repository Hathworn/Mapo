#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h-1)/stride + 1;
    int w = (in_w-1)/stride + 1;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int w_offset = (-size-1)/2 + 1;
    int h_offset = (-size-1)/2 + 1;
    int area = (size-1)/stride;

    int j = id % in_w;
    int i = (id / in_w) % in_h;
    int k = (id / (in_w * in_h)) % in_c;
    int b = id / (in_w * in_h * in_c);

    float d = 0.0f;
  
    // Unroll the loop to improve performance
    for (int l = -area; l <= area; ++l) {
        for (int m = -area; m <= area; ++m) {
            int out_w = (j - w_offset)/stride + m;
            int out_h = (i - h_offset)/stride + l;
            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h) {
                int out_index = out_w + w * (out_h + h * (k + in_c * b));
                d += (indexes[out_index] == id) ? delta[out_index] : 0.0f;
            }
        }
    }
    prev_delta[id] += d;
}