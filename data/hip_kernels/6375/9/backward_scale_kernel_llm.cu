#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int i, b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Pre-calculate offset for current filter
    int filter_offset = size * filter;

    // Iterate through batches
    for(b = 0; b < batch; ++b) {
        int batch_offset = size * n * b;

        // Iterate through elements with stride BLOCK
        for(i = p; i < size; i += BLOCK) {
            int index = i + filter_offset + batch_offset;
            sum += delta[index] * x_norm[index];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Use parallel reduction to accumulate sums
    for (i = BLOCK / 2; i > 0; i >>= 1) {
        if(p < i) part[p] += part[p + i];
        __syncthreads();
    }

    // Write the result of reduction to global memory
    if (p == 0) {
        atomicAdd(&scale_updates[filter], part[0]); // Use atomicAdd for reduction
    }
}