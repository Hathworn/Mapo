#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) return;

    // Pre-compute common multiplications outside the index calculations
    int stride_w1 = w1 * stride;
    int sample_w2 = w2 * sample;
    int h1_c1 = h1 * c1;
    int h2_c2 = h2 * c2;

    int i = id % minw;
    int j = (id / minw) % minh;
    int k = (id / (minw * minh)) % minc;
    int b = id / (minw * minh * minc);

    int out_index = i * sample + sample_w2 * j + h2_c2 * b + w2 * k;
    int add_index = i * stride + stride_w1 * j + h1_c1 * b + w1 * k;
    
    out[out_index] += add[add_index];
}