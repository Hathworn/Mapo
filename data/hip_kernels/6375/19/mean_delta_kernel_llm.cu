#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (i >= filters) return;

    float sum = 0.0f; // Local variable for accumulation

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += delta[index]; // Accumulate directly into local variable
        }
    }
    
    mean_delta[i] = sum * (-1.0f / sqrtf(variance[i] + 0.000001f)); // Use of sqrtf for float precision
}