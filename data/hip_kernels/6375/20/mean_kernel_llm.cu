#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.0f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Initialize shared memory for partial sums
    extern __shared__ float shared_mean[];
    shared_mean[threadIdx.x] = 0;

    // Calculate partial sums in each thread block
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            shared_mean[threadIdx.x] += x[index];
        }
    }

    // Sum up results across threads within a block
    __syncthreads();
    if (threadIdx.x == 0) {
        float sum = 0;
        for (int j = 0; j < blockDim.x; ++j) {
            sum += shared_mean[j];
        }
        mean[i] = sum * scale;
    }
}