#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void col2im_gpu_kernel(const int n, const float* data_col, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_im) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int num_threads = blockDim.x * gridDim.x;
    
    // Loop unrolling and handling multiple elements per thread
    for (; index < n; index += num_threads) {
        float val = 0;
        int w = index % width + pad;
        int h = (index / width) % height + pad;
        int c = index / (width * height);

        // Calculate output boundaries
        int w_col_start = max(0, (w < ksize) ? 0 : (w - ksize) / stride + 1);
        int w_col_end = min(w / stride + 1, width_col);
        int h_col_start = max(0, (h < ksize) ? 0 : (h - ksize) / stride + 1);
        int h_col_end = min(h / stride + 1, height_col);

        // Precompute offsets outside the loop
        int offset_base = (c * ksize * ksize + h * ksize + w) * height_col * width_col;
        int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
        int coeff_w_col = (1 - stride * height_col * width_col);

        // Use shared memory (if applicable) to reduce memory latency
        for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
            #pragma unroll // Unroll the inner loop for increased ILP
            for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
                val += data_col[offset_base + h_col * coeff_h_col + w_col * coeff_w_col];
            }
        }
        data_im[index] += val;
    }
}