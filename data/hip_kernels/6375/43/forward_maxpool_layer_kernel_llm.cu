#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, float *input, float *output, int *indexes)
{
    // Calculate output dimensions
    int h = (in_h-1)/stride + 1;
    int w = (in_w-1)/stride + 1;
    int c = in_c;

    // Calculate the unique global thread index for flattening
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if(id >= n) return;

    // Determine the positional indices in the output space
    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    int b = id / c;

    int w_offset = (-size-1)/2 + 1;
    int h_offset = (-size-1)/2 + 1;

    int out_index = j + w * (i + h * (k + c * b));
    float max = -INFINITY;
    int max_i = -1;

    // Optimize loop by unrolling
    for(int l = 0; l < size; ++l) {
        int cur_h = h_offset + i * stride + l;
        if (cur_h < 0 || cur_h >= in_h) continue;  // Early exit for invalid cur_h
        for(int m = 0; m < size; ++m) {
            int cur_w = w_offset + j * stride + m;
            if (cur_w < 0 || cur_w >= in_w) continue;  // Early exit for invalid cur_w
            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            float val = input[index];
            if(val > max) {
                max = val;
                max_i = index;
            }
        }
    }
    output[out_index] = max;
    indexes[out_index] = max_i;
}