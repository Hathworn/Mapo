#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Use shared memory for better performance if needed
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float dc_val = dc[i];
        float s_val = s[i];
        if (da) da[i] += dc_val * s_val;  // Combine loads and arithmetic
        db[i] += dc_val * (1 - s_val);   // Use stored value
        ds[i] += dc_val * (a[i] - b[i]); // Optimize arithmetic expression
    }
}