#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;
    
    // Ensure offset calculations are within valid range
    if(offset < size) {
        // Calculate the base index and then add the offset
        int index = (batch * n + filter) * size + offset;
        // Add bias to the corresponding output element
        output[index] += biases[filter];
    }
}