#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Use a 1D grid for improved performance
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we don't process out-of-bound indices
    if (i < N) {
        // Fetch the values first to improve memory access efficiency
        float x_val = X[i * INCX]; 
        float y_val = Y[i * INCY];
        Y[i * INCY] = y_val * x_val;
    }
}