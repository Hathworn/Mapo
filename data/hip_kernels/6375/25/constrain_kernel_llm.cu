#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access by using shared memory for ALPHA
__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(i < N) 
    {
        float value = X[i*INCX];
        X[i*INCX] = min(ALPHA, max(-ALPHA, value));
    }
}