#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Flatten block and thread indices

    // Use strided access to improve memory access pattern
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < N; i += stride) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}