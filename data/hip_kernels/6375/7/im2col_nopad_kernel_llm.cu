#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void im2col_nopad_kernel(float *im, int channels,  int height,  int width, int ksize,  int stride, float *data_col)
{
    // Calculate total number of threads and current thread index.
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate output dimensions for the col matrix.
    int height_col = (height - ksize) / stride + 1;
    int width_col = (width - ksize) / stride + 1;
    int channels_col = channels * ksize * ksize;
    
    // Compute the size of the column matrix once.
    int col_size = height_col * width_col * channels_col;
    
    // Exit if the current thread is beyond the col matrix bounds.
    if (id >= col_size) return;
    
    // Calculate indices for width, height, and channel.
    int col_index = id;
    int w = col_index % width_col;
    col_index /= width_col;
    int h = col_index % height_col;
    col_index /= height_col;
    int c = col_index % channels_col;
    
    // Precompute offsets and indices for efficient memory access.
    int w_offset = c % ksize;
    int h_offset = (c / ksize) % ksize;
    int im_channel = c / (ksize * ksize);
    int im_row = h_offset + h * stride;
    int im_col = w_offset + w * stride;
    
    // Use a conditional operator to determine the value to store.
    float val = (im_row >= 0 && im_col >= 0 && im_row < height && im_col < width) ? 
                im[im_col + width * (im_row + height * im_channel)] : 0.0f;
    
    data_col[id] = val;
}