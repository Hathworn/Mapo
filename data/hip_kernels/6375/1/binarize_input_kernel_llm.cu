#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = blockIdx.x * blockDim.x + threadIdx.x; // Adjusted indexing calculation 
    if (s >= size) return;

    // Initialize mean and accumulate using parallel reduction
    float mean = 0;
    for(int i = 0; i < n; ++i){
        mean += fabsf(input[i*size + s]); // Use fabsf for single precision
    }
    mean = mean / n;

    // Binarization
    for(int i = 0; i < n; ++i){
        binary[i*size + s] = (input[i*size + s] > 0) ? mean : -mean;
    }
}