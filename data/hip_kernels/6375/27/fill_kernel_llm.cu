#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized fill_kernel
__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplified linear index calculation
    int stride = gridDim.x * blockDim.x; // Calculate stride for looping over elements
    
    for (int i = idx; i < N; i += stride) { // Use loop to process multiple elements
        X[i*INCX] = ALPHA;
    }
}