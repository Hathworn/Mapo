#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[BLOCK];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    float sum = 0.0f;  // Initialize a local register for reduction

    for(int j = 0; j < batch; ++j)
    {
        for(int i = id; i < spatial; i += threads)
        {
            int index = j * spatial * filters + filter * spatial + i;
            sum += delta[index];  // Accumulate in the register
        }
    }

    local[id] = sum;  // Store the sum in shared memory
    __syncthreads();  // Synchronize to ensure all values are stored

    // Perform reduction in shared memory
    for(int stride = threads / 2; stride > 0; stride /= 2)
    {
        if(id < stride) 
        {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    if(id == 0)
    {
        mean_delta[filter] = local[0] * (-1.0f / sqrtf(variance[filter] + 0.000001f));
    }
}