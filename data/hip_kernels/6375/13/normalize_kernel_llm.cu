#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    // Calculate the global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds
    if (index >= N) return;

    // Calculate filter index
    int f = (index / spatial) % filters;

    // Pre-compute sqrt variance
    float inv_std_dev = rsqrtf(variance[f] + 0.000001f);

    // Normalize the input
    x[index] = (x[index] - mean[f]) * inv_std_dev;
}