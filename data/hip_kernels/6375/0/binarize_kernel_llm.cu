#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Use int for calculating grid position
    int totalThreads = gridDim.x * gridDim.y * blockDim.x;
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    // Optimize by processing multiple elements in each thread if possible
    for (; i < n; i += totalThreads) {
        binary[i] = (x[i] > 0) ? 1 : -1; // Simplified assignment
    }
}