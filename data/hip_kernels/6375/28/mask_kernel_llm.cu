#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified and more efficient index calculation
    int grid_size = gridDim.x * blockDim.x; // Total number of threads in the grid
    
    // Loop for processing more elements if grid is smaller than n
    for(; i < n; i += grid_size)
    {
        if(mask[i] == mask_num)
        {
            x[i] = mask_num;
        }
    }
}