#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use threadIdx.y for better access pattern
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = i * INCX;
    
    // Unrolling to improve performance
    for (; idx < N * INCX; idx += INCX * blockDim.x)
    {
        X[idx] *= ALPHA;
    }
}