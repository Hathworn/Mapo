#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure all threads are used efficiently using a grid-stride loop
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        Y[idx * INCY] = powf(X[idx * INCX], ALPHA); // Use powf for single precision
    }
}