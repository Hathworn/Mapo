#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    // Calculate unique thread index
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    // Use a register for local accumulation
    float local_sum = 0.0f;
    
    // Unroll loop partially to increase performance
    int k;
    for (k = 0; k < n; k += 4) {
        if (k < n) local_sum += x[k * groups + i];
        if ((k + 1) < n) local_sum += x[(k + 1) * groups + i];
        if ((k + 2) < n) local_sum += x[(k + 2) * groups + i];
        if ((k + 3) < n) local_sum += x[(k + 3) * groups + i];
    }
    
    // Write accumulated result back to global memory
    sum[i] = local_sum;
}