#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void im2col_pad_kernel(float *im, int channels, int height, int width, int ksize, int stride, float *data_col)
{
    int height_col = 1 + (height - 1) / stride;
    int width_col = 1 + (width - 1) / stride;
    int channels_col = channels * ksize * ksize;

    int pad = ksize / 2;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride_grid = gridDim.x * blockDim.x; // Calculate grid stride

    for (; id < height_col * width_col * channels_col; id += stride_grid) { // Loop through stride
        int col_index = id;
        int w = id % width_col;
        id /= width_col;
        int h = id % height_col;
        id /= height_col;
        int c = id % channels_col;

        int w_offset = c % ksize;
        int h_offset = (c / ksize) % ksize;
        int im_channel = c / ksize / ksize;
        int im_row = h_offset + h * stride - pad;
        int im_col = w_offset + w * stride - pad;

        // Use conditional operator to ensure bounds
        data_col[col_index] = (im_row >= 0 && im_col >= 0 && im_row < height && im_col < width) 
            ? im[im_col + width * (im_row + height * im_channel)] 
            : 0.0f;
    }
}