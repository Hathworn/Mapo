#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation
    int stride = blockDim.x * gridDim.x;  // Calculate stride for iteration
    for (; index < N; index += stride) {  // Iterate across all elements
        int f = (index / spatial) % filters;
        float inv_sqrt_var = 1.0f / (sqrt(variance[f]) + .000001f);  // Precompute 1/sqrt(variance[f])
        float two_spatial_batch = 2.0f / (spatial * batch);  // Precompute constant value

        delta[index] = delta[index] * inv_sqrt_var
                     + variance_delta[f] * two_spatial_batch * (x[index] - mean[f])
                     + mean_delta[f] / (spatial * batch);
    }
}