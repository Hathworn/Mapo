#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    int b = id / c;

    // Precompute and store parameters to reduce redundant calculations.
    int out_index = k + c * b;
    int offset = h * w * (k + b * c);
    float sum = 0.0f;

    // Use a local register variable 'sum' inside the loop for accumulation.
    for(int i = 0; i < w * h; ++i){
        sum += input[i + offset];
    }

    // Lazy evaluation enhancement: the calculation of division is done after loop completion.
    output[out_index] = sum / (w * h);
}