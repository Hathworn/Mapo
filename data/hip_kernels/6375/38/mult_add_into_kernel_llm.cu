#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify index calculation
    if (i < n) {
        c[i] += a[i] * b[i];  // Perform operation if within bounds
    }
}