#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Calculate scale factor outside of the loop
    float scale = 1.f / (batch * spatial - 1);

    // Use 1D grid and calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float variance_sum = 0.f; // Use a register for variance sum

    for(int j = 0; j < batch; ++j) {
        for(int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            float diff = x[index] - mean[i]; // Compute difference only once
            variance_sum += diff * diff;
        }
    }
    variance[i] = variance_sum * scale; // Use the local variance_sum
}