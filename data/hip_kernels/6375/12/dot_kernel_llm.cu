#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    // Use register variables for accumulation
    float sum = 0.0f, norm1 = 0.0f, norm2 = 0.0f;
    // Compute stride for batching
    int batchStride = size * n;

    for(int b = 0; b < batch; ++b){
        for(int i = 0; i < size; ++i){
            int i1 = b * batchStride + f1 * size + i;
            int i2 = b * batchStride + f2 * size + i;
            float val1 = output[i1];
            float val2 = output[i2];
            sum += val1 * val2;
            norm1 += val1 * val1;
            norm2 += val2 * val2;
        }
    }
    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    // Avoid calculating scale * sum / norm multiple times
    float scaleSumNorm = -scale * sum / norm;

    for(int b = 0; b < batch; ++b){
        for(int i = 0; i < size; ++i){
            int i1 = b * batchStride + f1 * size + i;
            int i2 = b * batchStride + f2 * size + i;
            // Minimize memory accesses by combining increments
            float val1 = output[i1];
            float val2 = output[i2];
            atomicAdd(&delta[i1], scaleSumNorm * val2);
            atomicAdd(&delta[i2], scaleSumNorm * val1);
        }
    }
}