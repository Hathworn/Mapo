#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0;

    // Ensure valid accesses and reduce index calculations
    for(int j = 0; j < batch; ++j) {
        for(int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += pow((x[index] - mean[filter]), 2);
        }
    }

    __syncthreads(); // Ensure all threads have written to local

    // Reduce local array using parallel reduction
    if (id == 0) {
        float sum = 0;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        variance[filter] = sum / (spatial * batch - 1);
    }
}