#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Reduce computation of grid index and check within valid range in the loop
    while (i < n) {
        c[i] = s[i] * a[i] + (1 - s[i]) * (b ? b[i] : 0);
        i += blockDim.x * gridDim.x; // Advance index by total number of threads
    }
}