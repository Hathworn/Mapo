#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void check_if_unique(const unsigned *keys, unsigned *is_unique, size_t kSize) {
    unsigned id = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;
    if (id < kSize) {
        // Simplified condition to initiate the uniqueness check
        is_unique[id] = (id == 0 || keys[id] != keys[id - 1]) ? 1 : 0;
    }
}