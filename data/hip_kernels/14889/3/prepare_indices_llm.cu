#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prepare_indices(const unsigned num_keys, unsigned *data) {
    // Use 2D indices for improved readability and optimization
    unsigned index = threadIdx.x + (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x;
    
    if (index < num_keys) {
        data[index] = index;
    }
}