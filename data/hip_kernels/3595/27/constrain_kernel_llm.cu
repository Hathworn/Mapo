#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if(i < N) {
        float value = X[i*INCX];
        value = fminf(ALPHA, value); // Use temporary variable to store result
        value = fmaxf(-ALPHA, value);
        X[i*INCX] = value; // Write back result
    }
}