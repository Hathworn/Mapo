#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;
    
    // Calculate filter index
    int f = (index / spatial) % filters;
    
    // Precompute constants for use in the main computation
    float inv_sqrt_var = 1.f / sqrtf(variance[f] + .00001f);
    float two_var_delta = variance_delta[f] * 2.f / (spatial * batch);
    float mean_delta_scaled = mean_delta[f] / (spatial * batch);
    
    // Apply the normalization and update delta
    delta[index] = delta[index] * inv_sqrt_var + two_var_delta * (x[index] - mean[f]) + mean_delta_scaled;
}