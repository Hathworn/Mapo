#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeHessianListS0(float *trans_x, float *trans_y, float *trans_z, int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *centroid_x, double *centroid_y, double *centroid_z, double *icov00, double *icov01, double *icov02, double *icov10, double *icov11, double *icov12, double *icov20, double *icov21, double *icov22, double *point_gradients, double *tmp_hessian, int valid_voxel_num)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x; 
    int stride = blockDim.x * gridDim.x; 
    int col = blockIdx.y; 

    if (col < 6) {
        double *tmp_pg0 = point_gradients + col * valid_points_num;
        double *tmp_pg1 = tmp_pg0 + 6 * valid_points_num;
        double *tmp_pg2 = tmp_pg1 + 6 * valid_points_num;
        double *tmp_h = tmp_hessian + col * valid_voxel_num;

        // Use a shared memory buffer for centroids and inverse covariance
        __shared__ double s_centroid_x[256];
        __shared__ double s_centroid_y[256];
        __shared__ double s_centroid_z[256];
        __shared__ double s_icov00[256];
        __shared__ double s_icov01[256];
        __shared__ double s_icov02[256];
        __shared__ double s_icov10[256];
        __shared__ double s_icov11[256];
        __shared__ double s_icov12[256];
        __shared__ double s_icov20[256];
        __shared__ double s_icov21[256];
        __shared__ double s_icov22[256];

        for (int i = id; i < valid_points_num; i += stride) {
            int pid = valid_points[i];
            double d_x = static_cast<double>(trans_x[pid]);
            double d_y = static_cast<double>(trans_y[pid]);
            double d_z = static_cast<double>(trans_z[pid]);

            double pg0 = tmp_pg0[i];
            double pg1 = tmp_pg1[i];
            double pg2 = tmp_pg2[i];

            for (int j = starting_voxel_id[i]; j < starting_voxel_id[i + 1]; j++) {
                int vid = voxel_id[j];
                
                // Load data to shared memory (with bounds check)
                if (threadIdx.x < 256) {
                    s_centroid_x[threadIdx.x] = centroid_x[vid];
                    s_centroid_y[threadIdx.x] = centroid_y[vid];
                    s_centroid_z[threadIdx.x] = centroid_z[vid];
                    s_icov00[threadIdx.x] = icov00[vid];
                    s_icov01[threadIdx.x] = icov01[vid];
                    s_icov02[threadIdx.x] = icov02[vid];
                    s_icov10[threadIdx.x] = icov10[vid];
                    s_icov11[threadIdx.x] = icov11[vid];
                    s_icov12[threadIdx.x] = icov12[vid];
                    s_icov20[threadIdx.x] = icov20[vid];
                    s_icov21[threadIdx.x] = icov21[vid];
                    s_icov22[threadIdx.x] = icov22[vid];
                }
                __syncthreads();
                
                // Use shared memory
                tmp_h[j] = (d_x - s_centroid_x[threadIdx.x]) * (s_icov00[threadIdx.x] * pg0 + s_icov01[threadIdx.x] * pg1 + s_icov02[threadIdx.x] * pg2)
                         + (d_y - s_centroid_y[threadIdx.x]) * (s_icov10[threadIdx.x] * pg0 + s_icov11[threadIdx.x] * pg1 + s_icov12[threadIdx.x] * pg2)
                         + (d_z - s_centroid_z[threadIdx.x]) * (s_icov20[threadIdx.x] * pg0 + s_icov21[threadIdx.x] * pg1 + s_icov22[threadIdx.x] * pg2);
            }
        }
    }
}