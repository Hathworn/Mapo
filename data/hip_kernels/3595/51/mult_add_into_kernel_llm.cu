#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Use shared memory to efficiently handle access
    __shared__ float shared_a[256];  // Assumes blockDim.x <= 256
    __shared__ float shared_b[256];  // Assumes blockDim.x <= 256

    // Load data into shared memory
    if(i < n) {
        shared_a[threadIdx.x] = a[i];
        shared_b[threadIdx.x] = b[i];
        __syncthreads();

        // Perform computation using shared memory
        c[i] += shared_a[threadIdx.x] * shared_b[threadIdx.x];
    }
}