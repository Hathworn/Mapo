#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void wgan_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate the global thread index more concisely
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    // Use a single condition to check if index is within bounds and execute operation
    if(i < n) {
        float t = truth[i];
        float p = pred[i];
        error[i] = t ? -p : p;
        delta[i] = t > 0 ? 1 : -1;
    }
}