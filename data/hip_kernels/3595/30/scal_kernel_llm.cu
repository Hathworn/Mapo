#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Optimize thread index calculation and reduce operations
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x; 
    if(i < N)
    {
        X[i * INCX] *= ALPHA; 
    }
}