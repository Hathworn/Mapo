#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce repetitive computations
    extern __shared__ float shmem[];
    float *s_truth = shmem;
    float *s_pred = &shmem[blockDim.x];

    if (i < n) {
        s_truth[threadIdx.x] = truth[i];
        s_pred[threadIdx.x] = pred[i];
        __syncthreads(); // Ensure all threads have loaded data

        float t = s_truth[threadIdx.x];
        float p = s_pred[threadIdx.x];
        error[i] = -t * logf(p + 1e-7f) - (1 - t) * logf(1 - p + 1e-7f); // Use logf for float operations
        delta[i] = t - p;
    }
}