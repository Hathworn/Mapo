#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Use a 2D grid and 2D block to optimize memory access
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int index = idy * blockDim.x * gridDim.x + idx;
    if(index >= N) return;

    int in_index = index;
    int in_w = index % w;
    index /= w;
    int in_h = index % h;
    index /= h;
    int in_c = index % c;
    index /= c;
    int b = index % batch;

    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    // Use shared memory for temporary storage if needed
    if(forward) 
        out[out_index] = x[in_index];
    else 
        out[in_index] = x[out_index];
}