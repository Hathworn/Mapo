#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index in one step for better performance
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Ensure the loop only iterates required times using CUDA's built-in index handling
    if (i * INCX < N) {
        X[i * INCX] = ALPHA;
    }
}