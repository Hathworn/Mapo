#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output) {
    float sum = 0;
    float largest = -INFINITY;
    // Combine the two loops to improve memory access patterns
    for (int i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = (val > largest) ? val : largest;
        float e = expf(val / temp - largest / temp);
        sum += e;
        output[i * stride] = e;
    }
    // Perform normalization in separate loop for coalesced memory access
    for (int i = 0; i < n; ++i) {
        output[i * stride] /= sum;
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Early exit for threads that have no work to do
    if (id >= batch * groups) return;
    int b = id / groups;
    int g = id % groups;
    // Launch device function for individual work
    softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}