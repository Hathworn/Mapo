#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int stride = gridDim.x * blockDim.x; // Calculate grid stride

    for (int i = idx; i < N; i += stride) { // Loop with stride to cover all elements
        Y[i * INCY] = powf(X[i * INCX], ALPHA); // Use powf for improved performance with float
    }
}