#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Compute global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int size = gridDim.x * blockDim.x;

    // Loop over elements with stride equal to total number of threads
    for (; i < N; i += size) {
        Y[i * INCY] *= X[i * INCX];
    }
}