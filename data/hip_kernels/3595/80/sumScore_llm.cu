#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumScore(double *score, int full_size, int half_size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory for better performance
    extern __shared__ double sharedScore[];

    for (int i = index; i < half_size; i += stride) {
        sharedScore[threadIdx.x] = score[i];
        __syncthreads();
        
        if (i + half_size < full_size) {
            sharedScore[threadIdx.x] += score[i + half_size];
        }
        
        __syncthreads();
        score[i] = sharedScore[threadIdx.x];
    }
}