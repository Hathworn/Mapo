#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    // Calculate global index
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Precompute stride dimensions
    int stride_w = w * stride;
    int stride_h = h * stride;

    // Calculate output indices efficiently
    int out_index = i;
    int out_w = i % stride_w;
    i /= stride_w;
    int out_h = i % stride_h;
    i /= stride_h;
    int out_c = i % c;
    i /= c;
    int b = i % batch;

    // Calculate input indices efficiently
    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_c = out_c;

    int in_index = b * w * h * c + in_c * w * h + in_h * w + in_w;

    // Perform the operation based on the forward flag
    if (forward) {
        out[out_index] += scale * x[in_index];  // Simplified expression
    } else {
        atomicAdd(x + in_index, scale * out[out_index]);  // Simplified expression
    }
}