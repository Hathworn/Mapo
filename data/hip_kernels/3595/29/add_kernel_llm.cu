#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the unique thread index within the grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride to efficiently process elements beyond the first block
    for (; i < N; i += blockDim.x * gridDim.x)
    {
        // Perform the addition operation on the indexed element
        X[i*INCX] += ALPHA;
    }
}