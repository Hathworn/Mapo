#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use loop to allow each thread to handle multiple elements if N is large
    for (int i = index; i < N; i += stride) {
        // Pre-compute factors outside of the loop for better performance
        float bias_correction1 = 1.f - powf(B1, t);
        float bias_correction2 = 1.f - powf(B2, t);

        float mhat = m[i] / bias_correction1;
        float vhat = v[i] / bias_correction2;

        x[i] = x[i] + rate * mhat / (sqrtf(vhat) + eps);
    }
}