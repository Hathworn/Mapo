#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Optimize by calculating the effective index once
    int i = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x;

    // Check bounds of i
    if (i < (NX + NY) * B) {
        int b = i / (NX + NY);
        int j = i % (NX + NY);

        // Reduce branching by checking X or Y presence outside condition
        if (j < NX) {
            if (X) {
                X[b * NX + j] += OUT[i];
            }
        } else {
            if (Y) {
                Y[b * NY + j - NX] += OUT[i];
            }
        }
    }
}