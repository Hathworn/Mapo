#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    // Calculate unique thread index across grid
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Compute stride for efficient grid-stride loop
    for (int i = index; i < N; i += stride) { 
        int f = (i / spatial) % filters; // Calculate filter index
        x[i] = (x[i] - mean[f]) / (sqrtf(variance[f] + .00001f)); // Normalize
    }
}