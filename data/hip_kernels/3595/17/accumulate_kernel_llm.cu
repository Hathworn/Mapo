#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Compute unique index for each thread
    if (i >= groups) return;  // Early exit for threads beyond the range

    float local_sum = 0.0f;  // Use a local variable to store sum
    for (int k = 0; k < n; ++k) {
        local_sum += x[k * groups + i];  // Accumulate values from global memory
    }
    sum[i] = local_sum;  // Write the computed sum to global memory
}