#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    // Calculate global thread ID.
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x; 
    if(id >= n) return;

    // Compute indices for dimensions.
    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    int w_offset = -(size / 2);
    int h_offset = -(size / 2);

    int out_index = j + w * (i + h * (k + c * b));

    // Inlined offset calculation and reduced condition checks.
    for(int l = 0; l < size; ++l) {
        int cur_h = h_offset + i + l;
        if(cur_h < 0 || cur_h >= h) continue; // Skip when out of bounds.

        for(int m = 0; m < size; ++m) {
            int cur_w = w_offset + j + m;
            if(cur_w < 0 || cur_w >= w) continue; // Skip when out of bounds.
            
            int index = cur_w + w * (cur_h + h * (k + b * c));
            delta[out_index] += rate * (x[index] - x[out_index]);
        }
    }
}