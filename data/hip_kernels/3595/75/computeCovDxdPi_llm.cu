#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeCovDxdPi(int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *inverse_covariance, int voxel_num, double gauss_d1, double gauss_d2, double *point_gradients, double *cov_dxd_pi, int valid_voxel_num)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int row = blockIdx.y;
    int col = blockIdx.z;

    // Ensure that threads within bounds execute the computation
    if (row < 3 && col < 6) {
        double *icov0 = inverse_covariance + row * 3 * voxel_num;
        double *icov1 = icov0 + voxel_num;
        double *icov2 = icov1 + voxel_num;
        double *cov_dxd_pi_tmp = cov_dxd_pi + (row * 6 + col) * valid_voxel_num;
        double *pg_tmp0 = point_gradients + col * valid_points_num;
        double *pg_tmp1 = pg_tmp0 + 6 * valid_points_num;
        double *pg_tmp2 = pg_tmp1 + 6 * valid_points_num;

        // Using shared memory for point_gradients to reduce redundant loads
        __shared__ double shared_pg[3];
        
        for (int i = id; i < valid_points_num; i += stride) {
            // Load point gradients into shared memory
            shared_pg[0] = pg_tmp0[i];
            shared_pg[1] = pg_tmp1[i];
            shared_pg[2] = pg_tmp2[i];
            __syncthreads();

            for (int j = starting_voxel_id[i]; j < starting_voxel_id[i + 1]; j++) {
                int vid = voxel_id[j];

                // Using shared memory to access already loaded gradients
                cov_dxd_pi_tmp[j] = icov0[vid] * shared_pg[0] + icov1[vid] * shared_pg[1] + icov2[vid] * shared_pg[2];
            }
        }
    }
}