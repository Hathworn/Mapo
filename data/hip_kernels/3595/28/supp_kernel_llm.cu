#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Iterate through the data chunck by stride jumping
    for (; i < N; i += blockDim.x * gridDim.x) {
        // Pre-compute common expression
        int index = i * INCX;
        float value = X[index] * X[index];
        
        // Conditional update to zero
        if(value < (ALPHA * ALPHA)) {
            X[index] = 0;
        }
    }
}