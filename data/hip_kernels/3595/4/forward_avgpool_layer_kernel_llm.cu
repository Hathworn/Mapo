#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 256

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    // Calculate global index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;
    
    // Determine batch index and channel index
    int k = id % c;
    int b = id / c;
    
    // Precompute indices
    int out_index = k + c * b;
    int in_base_index = h * w * (k + b * c);
    
    // Initialize output to zero
    float sum = 0.0f;
    
    // Accumulate input values
    for(int i = 0; i < w * h; ++i){
        sum += input[in_base_index + i];
    }
    
    // Compute average
    output[out_index] = sum / (w * h);
}
```
