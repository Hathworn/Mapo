#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    // Calculate output dimensions
    int h = (in_h + 2*pad)/stride;
    int w = (in_w + 2*pad)/stride;
    
    // Compute the search area size
    int area = (size-1)/stride;

    // Calculate unique global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    // Decode index into batch (b), channel (k), height (i), width (j)
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    // Offset for padding
    int w_offset = -pad;
    int h_offset = -pad;

    float d = 0;

    // Iterate over neighborhood area
    for(int l = -area; l <= area; ++l){
        for(int m = -area; m <= area; ++m){
            // Calculate output indices
            int out_w = (j-w_offset)/stride + m;
            int out_h = (i-h_offset)/stride + l;
            int out_index = out_w + w * (out_h + h * (k + in_c * b));

            // Check validity and accumulate delta
            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h && indexes[out_index] == id) {
                d += delta[out_index];
            }
        }
    }

    // Update previous layer's delta
    atomicAdd(&prev_delta[id], d);
}