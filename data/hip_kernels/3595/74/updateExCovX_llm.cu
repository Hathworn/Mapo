#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateExCovX(double *e_x_cov_x, double gauss_d2, int valid_voxel_num)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Process each element in the range of this thread
    for (int i = id; i < valid_voxel_num; i += stride) {
        e_x_cov_x[i] *= gauss_d2;
    }
}