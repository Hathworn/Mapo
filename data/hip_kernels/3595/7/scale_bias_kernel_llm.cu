#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;
    
    // Ensure the index is within boundary
    if(offset < size) {
        // Use shared memory to cache biases for better performance
        __shared__ float shared_biases[512];
        
        if (threadIdx.x == 0) {
            shared_biases[filter] = biases[filter];
        }
        __syncthreads();
        
        // Perform the operation
        output[(batch * n + filter) * size + offset] *= shared_biases[filter];
    }
}