#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation assuming 1D grid configuration

    // Utilize shared memory or registers to minimize global memory accesses 
    if (i < n) {
        float si = s[i];
        float dci = dc[i];
        if (da) da[i] += dci * si;
        if (db) db[i] += dci * (1 - si);
        ds[i] += dci * (a[i] - b[i]);
    }
}