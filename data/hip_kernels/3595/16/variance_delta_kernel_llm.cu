#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float temp_variance = -.5f * powf(variance[i] + .00001f, (float)(-3.f/2.f));
    float sum = 0;

    // Optimize memory access by using a single loop
    for (int j = 0; j < batch * spatial; ++j) {
        int index = j * filters + i;
        sum += delta[index] * (x[index] - mean[i]);
    }

    variance_delta[i] = sum * temp_variance;
}