#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    // Calculate a global unique index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    // Calculate corresponding batch and spatial indices
    int b = index / spatial;
    int i = index % spatial;
    int f;
    float sum = 0.0f;

    // Use shared memory for reducing memory operations
    extern __shared__ float shared_mem[];
    float* local_x = shared_mem;

    // Accumulate squared values in shared memory
    for(f = 0; f < filters; ++f){
        int global_index = b*filters*spatial + f*spatial + i;
        local_x[f] = x[global_index];
        sum += local_x[f] * local_x[f];
    }
    sum = sqrtf(sum);
    if(sum == 0.0f) sum = 1.0f;

    // Normalize in shared memory and update global memory
    for(f = 0; f < filters; ++f){
        int global_index = b*filters*spatial + f*spatial + i;
        x[global_index] = local_x[f] / sum;
        dx[global_index] = (1.0f - x[global_index]) / sum;
    }
}