#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Compute global index based on block and thread indices
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Ensure index is less than the total number of elements
    if(i < n) {
        // Perform weighted sum calculation
        c[i] = s[i] * a[i] + (1.0f - s[i]) * (b ? b[i] : 0.0f);
    }
}