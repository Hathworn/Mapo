#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure id is within bounds using a loop rather than a guard clause for efficiency
    while (id < size) {
        int i = id % minw;
        id /= minw;
        int j = id % minh;
        id /= minh;
        int k = id % minc;
        id /= minc;
        int b = id % batch;

        int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
        int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));
        out[out_index] = s1 * out[out_index] + s2 * add[add_index];

        id += blockDim.x * gridDim.x; // Move to next id up to total number of threads
    }
}