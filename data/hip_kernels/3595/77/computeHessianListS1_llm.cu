#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeHessianListS1(float *trans_x, float *trans_y, float *trans_z, int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *centroid_x, double *centroid_y, double *centroid_z, double gauss_d1, double gauss_d2, double *hessians, double *e_x_cov_x, double *tmp_hessian, double *cov_dxd_pi, double *point_gradients, int valid_voxel_num)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int row = blockIdx.y;
    int col = blockIdx.z;

    if (row < 6 && col < 6) {
        // Optimize pointer arithmetic by combining calculations
        double *cov_dxd_pi_base = cov_dxd_pi + row * valid_voxel_num;
        double *tmp_h = tmp_hessian + col * valid_voxel_num;
        double *h = hessians + (row * 6 + col) * valid_points_num;
        double *tmp_pg_base = point_gradients + col * valid_points_num;

        for (int i = id; i < valid_points_num; i += stride) {
            int pid = valid_points[i];
            double d_x = static_cast<double>(trans_x[pid]);
            double d_y = static_cast<double>(trans_y[pid]);
            double d_z = static_cast<double>(trans_z[pid]);

            // Use registers for frequently accessed values
            double pg0 = tmp_pg_base[i];
            double pg1 = tmp_pg_base[i + valid_points_num * 6];
            double pg2 = tmp_pg_base[i + valid_points_num * 12];

            double final_hessian = 0.0;

            for (int j = starting_voxel_id[i]; j < starting_voxel_id[i + 1]; j++) {
                int vid = voxel_id[j];
                double tmp_ex = e_x_cov_x[j];

                if (!(tmp_ex > 1 || tmp_ex < 0 || tmp_ex != tmp_ex)) {
                    // Unroll cov_dxd calculations and reduce repetitive indexing
                    double cov_dxd0 = cov_dxd_pi_base[j];
                    double cov_dxd1 = cov_dxd_pi_base[j + valid_voxel_num * 6];
                    double cov_dxd2 = cov_dxd_pi_base[j + valid_voxel_num * 12];

                    tmp_ex *= gauss_d1;

                    // Fuse operations to minimize instruction count
                    final_hessian += tmp_ex * (-gauss_d2 * ((d_x - centroid_x[vid]) * cov_dxd0 + (d_y - centroid_y[vid]) * cov_dxd1 + (d_z - centroid_z[vid]) * cov_dxd2) * tmp_h[j] + (pg0 * cov_dxd0 + pg1 * cov_dxd1 + pg2 * cov_dxd2));
                }
            }

            h[i] = final_hessian;
        }
    }
}