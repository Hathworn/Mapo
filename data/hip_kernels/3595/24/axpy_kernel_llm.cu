#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use a 1D grid for simplicity
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index is within bounds
    if (i < N) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX]; // Perform the operation
    }
}
```
