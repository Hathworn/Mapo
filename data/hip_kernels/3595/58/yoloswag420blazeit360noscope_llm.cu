#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    // Use blockDim.x and gridDim.x to calculate thread id more efficiently
    int id = blockIdx.x * blockDim.x + threadIdx.x; 
    
    if(id < size) input[id] *= (rand[id] >= prob) * scale; // Use branchless condition
}