#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Combine x and y grid indices into a single x index
    int stride = gridDim.x * blockDim.x; // Calculate stride once

    for(int i = idx; i < N; i += stride) { // Use strided loop for better memory access
        int in_s = i % spatial;
        int in_c = (i / spatial) % layers;
        int b = i / (spatial * layers);

        int i1 = b * layers * spatial + in_c * spatial + in_s;
        int i2 = b * layers * spatial + in_s * layers + in_c;

        if (forward) out[i2] = x[i1];
        else out[i1] = x[i2];
    }
}