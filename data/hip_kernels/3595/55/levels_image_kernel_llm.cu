#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int size = batch * w * h;
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;

    int x = id % w;
    int y = (id / w) % h;
    int z = id / (w * h);

    // Pre-calculate index for accessing image array
    size_t offset = (z * h + y) * w * 3 + x;
    float r = image[offset];
    float g = image[offset + w * h];
    float b = image[offset + 2 * w * h];
    
    float rshift = rand[0];
    float gshift = rand[1];
    float bshift = rand[2];
    float r0 = rand[8 * z];
    float r1 = rand[8 * z + 1];
    float r2 = rand[8 * z + 2];
    float r3 = rand[8 * z + 3];

    saturation = r0 * (saturation - 1) + 1;
    saturation = (r1 > .5f) ? 1.f / saturation : saturation;
    exposure = r2 * (exposure - 1) + 1;
    exposure = (r3 > .5f) ? 1.f / exposure : exposure;

    float3 rgb = make_float3(r, g, b);

    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }

    image[offset] = rgb.x * scale + translate + (rshift - 0.5f) * shift;
    image[offset + w * h] = rgb.y * scale + translate + (gshift - 0.5f) * shift;
    image[offset + 2 * w * h] = rgb.z * scale + translate + (bshift - 0.5f) * shift;
}