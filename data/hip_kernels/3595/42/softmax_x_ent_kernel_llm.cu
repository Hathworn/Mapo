#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < n) {
        float t = truth[i];
        float p = pred[i];
        error[i] = (t > 0.0f) ? -logf(p) : 0.0f; // Use logf for float precision
        delta[i] = t - p;
    }
}