#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for processing larger arrays

    // Loop to handle cases where total number of threads is less than n
    for (; i < n; i += stride)
    {
        if (mask[i] == mask_num)
        {
            x[i] = val;
        }
    }
}