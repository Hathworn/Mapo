#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel function by using blockIdx.x instead of calculating a linear index manually.
__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n){
        float diff = truth[i] - pred[i];
        error[i] = diff * diff;
        delta[i] = diff;
    }
}