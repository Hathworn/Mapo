#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeScoreList(int *starting_voxel_id, int *voxel_id, int valid_points_num, double *e_x_cov_x, double gauss_d1, double *score)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    if (id < valid_points_num) {  // Bound check for threads
        double score_inc = 0;

        for (int vid = starting_voxel_id[id]; vid < starting_voxel_id[id + 1]; vid++) {
            double tmp_ex = e_x_cov_x[vid];

            // Simplified condition check using isnan for NaN values
            if (tmp_ex >= 0 && tmp_ex <= 1) {
                score_inc += -gauss_d1 * tmp_ex;
            }
        }

        score[id] = score_inc;
    }
}