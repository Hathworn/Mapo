#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    // Calculate unique thread index with better optimization
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x; // Total number of threads

    // Iterate over elements processed by each thread to improve workload distribution
    for (int index = idx; index < n * size * batch; index += total_threads) {
        int i = index % size;
        int j = (index / size) % n;
        int k = (index / (size * n));

        output[(k * n + j) * size + i] += biases[j];
    }
}