#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary) 
{
    // Combine block and thread indices to a single linear index
    int s = blockIdx.x * blockDim.x + threadIdx.x; 
    if (s >= size) return; // Early exit if out of bounds

    float mean = 0;
    
    // Use shared memory to improve access time
    __shared__ float shared_mean;
    if (threadIdx.x == 0) shared_mean = 0;

    // Reduce unnecessary global memory access by calculating mean within block
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        atomicAdd(&shared_mean, fabsf(input[i * size + s]));
    }
    __syncthreads();

    mean = shared_mean / n;
    
    // Binirization operation
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}