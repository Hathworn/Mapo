#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.f / (batch * spatial);
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Use shared memory for better memory access efficiency
    __shared__ float shared_mean[256];
    shared_mean[threadIdx.x] = 0;  // Initialize shared memory
    __syncthreads();  // Synchronize to ensure all threads have initialized
    
    int j, k;
    for (j = 0; j < batch; ++j) {
        for (k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            shared_mean[threadIdx.x] += x[index];  // Accumulate values in shared memory
        }
    }
    __syncthreads();  // Synchronize to ensure all accumulations are complete

    if (threadIdx.x == 0) {
        mean[i] = 0;
        for (int t = 0; t < min(blockDim.x, filters - i); ++t) {
            mean[i] += shared_mean[t];  // Reduce in block
        }
        mean[i] *= scale;  // Apply scale factor
    }
}