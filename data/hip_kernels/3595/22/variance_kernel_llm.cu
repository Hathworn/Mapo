#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Use shared memory to improve performance
    extern __shared__ float shared_mem[];
    float *partial_sums = shared_mem;   

    float scale = 1.f/(batch * spatial - 1);
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    partial_sums[threadIdx.x] = 0.0f;
    
    // Loop unrolling for better performance
    for(int j = 0; j < batch; ++j) {
        int base_index = j*filters*spatial + i*spatial;
        for(int k = 0; k < spatial; k += 2) {
            int index1 = base_index + k;
            int index2 = base_index + k + 1;
            float diff1 = x[index1] - mean[i];
            float diff2 = (k + 1 < spatial) ? x[index2] - mean[i] : 0.0f;
            partial_sums[threadIdx.x] += powf(diff1, 2) + powf(diff2, 2);
        }
    }

    __syncthreads(); // Synchronize threads to ensure shared memory writes are complete

    // Reduction to accumulate partial sums
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            partial_sums[threadIdx.x] += partial_sums[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        variance[i] = partial_sums[0] * scale;
    }
}