#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output) {
    int i;
    float sum = 0;
    float largest = -INFINITY;

    // Unroll loop to improve performance
    for (i = 0; i < n; ++i) {
        int val = input[i * stride];
        largest = (val > largest) ? val : largest;
    }

    // Combine computations to reduce loop overhead and improve performance
    for (i = 0; i < n; ++i) {
        float e = expf(input[i * stride] / temp - largest / temp);
        sum += e;
        output[i * stride] = e;
    }

    // Combine normalization into previous loop to avoid extra pass
    for (i = 0; i < n; ++i) {
        output[i * stride] /= sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;

    // Precompute indices to improve performance
    int s = id % spatial;
    id /= spatial;
    int g = id % groups;
    int b = id / groups;
    int goff = group_offset[g] * spatial;
    int boff = b * stride;

    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}