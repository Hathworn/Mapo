#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel using blockIdx and threadIdx
__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int step = gridDim.x * blockDim.x;
    for (; i < n; i += step) {
        binary[i] = (x[i] >= 0) ? 1.0f : -1.0f; // Loop for larger arrays
    }
}