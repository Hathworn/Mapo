#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified 1D grid and block indexing
    int stride = gridDim.x * blockDim.x;           // Calculate stride for effective processing
    for (; i < n; i += stride) {                   // Loop over elements with stride
        if (mask[i] == mask_num) {                 // Apply mask condition
            x[i] *= scale;                         // Scale the value
        }
    }
}