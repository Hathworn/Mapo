#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate the global index for the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Operate only within valid range
    if (i < (NX + NY) * B) {
        int b = i / (NX + NY);
        int j = i % (NX + NY);
        
        // Use conditional operator for concise array access
        OUT[i] = (j < NX) ? X[b * NX + j] : Y[b * NY + j - NX];
    }
}