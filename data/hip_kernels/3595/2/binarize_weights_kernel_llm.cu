#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified linear index calculation
    if (f >= n) return;
    
    float mean = 0.0f;
    int offset = f * size;  // Precompute offset for the current feature map

    // Use a single loop to avoid repeated calculation of offset
    for(int i = 0; i < size; ++i){
        float weight = weights[offset + i];
        mean += fabsf(weight);
    }
    mean = mean / size;
    
    for(int i = 0; i < size; ++i){
        float weight = weights[offset + i];
        binary[offset + i] = (weight > 0) ? mean : -mean;
    }
}