#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Use shared memory for faster access
    extern __shared__ float shared_mem[];
    
    int thread_id = threadIdx.x;
    int global_id = blockIdx.x * blockDim.x + thread_id;
    
    // Load data into shared memory
    if(global_id < n) {
        shared_mem[thread_id] = truth[global_id] - pred[global_id];
    }
    __syncthreads();
    
    if(global_id < n) {
        float diff = shared_mem[thread_id];
        float abs_val = fabsf(diff);
        if(abs_val < 1.0f) {
            error[global_id] = diff * diff;
            delta[global_id] = diff;
        }
        else {
            error[global_id] = 2.0f * abs_val - 1.0f;
            delta[global_id] = copysignf(1.0f, diff);
        }
    }
}