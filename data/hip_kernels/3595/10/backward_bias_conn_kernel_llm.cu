#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;  // Use 1D indexing for better parallel efficiency
    if (index >= n) return;
    
    float sum = 0.0f;  // Explicit float type
    for(int b = 0; b < batch; ++b){
        int i = b * n + index;
        sum += delta[i];
    }
    
    atomicAdd(&bias_updates[index], sum); // Use atomic addition to avoid race conditions
}