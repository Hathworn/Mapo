#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Use 1D grid and block indexing
    if(idx >= n * w * h * c) return;  // Increase parallel work

    int i = idx % (w * h);  // Calculate input index
    int k = (idx / (w * h)) % c;
    int b = idx / (w * h * c);

    int out_index = k + c * b;  // Calculate output index
    int in_index = i + w * h * (k + b * c);
    in_delta[in_index] += out_delta[out_index] / (w * h);  // Divide once instead of in loop
}