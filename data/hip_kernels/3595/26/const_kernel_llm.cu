#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Stride loop to handle large arrays
    int stride = gridDim.x * blockDim.x;
    for (; i < N; i += stride)
    {
        X[i * INCX] = ALPHA; // Assign ALPHA to each element
    }
}