#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computePointHessian1(float *x, float *y, float *z, int points_num, int *valid_points, int valid_points_num, double *dh_ang, double *ph124, double *ph134, double *ph144, double *ph154, double *ph125, double *ph164, double *ph135, double *ph174, double *ph145)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    __shared__ double h_ang[18];

    // Load shared memory once per block
    if (threadIdx.x < 18) {
        h_ang[threadIdx.x] = dh_ang[18 + threadIdx.x];
    }

    __syncthreads();

    // Process points in stride
    for (int i = id; i < valid_points_num; i += stride) {
        int pid = valid_points[i];

        // Load coordinates and convert to double only once
        double o_x = static_cast<double>(x[pid]);
        double o_y = static_cast<double>(y[pid]);
        double o_z = static_cast<double>(z[pid]);

        // Use intermediate variables to reduce redundant computations
        double p1 = o_x * h_ang[9] + o_y * h_ang[10] + o_z * h_ang[11];
        double p2 = o_x * h_ang[12] + o_y * h_ang[13] + o_z * h_ang[14];
        double p3 = o_x * h_ang[15] + o_y * h_ang[16] + o_z * h_ang[17];

        ph124[i] = o_x * h_ang[0] + o_y * h_ang[1] + o_z * h_ang[2];
        ph134[i] = o_x * h_ang[3] + o_y * h_ang[4] + o_z * h_ang[5];
        ph144[i] = o_x * h_ang[6] + o_y * h_ang[7] + o_z * h_ang[8];

        ph154[i] = ph125[i] = p1;
        ph164[i] = ph135[i] = p2;
        ph174[i] = ph145[i] = p3;
    }
}