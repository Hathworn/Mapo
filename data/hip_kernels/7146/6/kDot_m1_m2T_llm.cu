#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDot_m1_m2T(const int nThreads, const float *m1, const float *m2, float *output, const int m1_columns, const int m2_rows){
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nThreads) {  // Ensure not to exceed nThreads
        int r = i / m2_rows; // Calculate row
        int c = i % m2_rows; // Calculate column
        float t_output = 0.0;
        int id_T;

        // Use shared memory to load m2 rows for improved efficiency
        __shared__ float m2_shared[1024]; // Adjust size as required
        for (int k = 0; k < m1_columns; ++k) {
            if (threadIdx.x < m1_columns) {
                m2_shared[k] = m2[c * m1_columns + threadIdx.x];
            }
            __syncthreads(); // Synchronize threads within block

            t_output += m1[r * m1_columns + k] * m2_shared[k];

            __syncthreads(); // Ensures all reads are done before reload
        }

        output[i] = t_output; // Write result to output
    }
}