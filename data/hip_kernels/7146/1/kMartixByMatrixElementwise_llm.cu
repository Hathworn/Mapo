#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMartixByMatrixElementwise(const int nThreads, const float *m1, const float *m2, float *output) {
    // Optimize accessing memory by minimizing thread divergence and improving coalesced memory access
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nThreads) {
        output[i] = m1[i] * m2[i];  // Perform elementwise multiplication
    }
}