#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSigmoid(const int nThreads, float const *input, float *output) {
    /* Compute the value of the sigmoid function f(x) = 1/(1 + e^-x) efficiently */
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (; i < nThreads; i += stride) {
        output[i] = 1.0f / (1.0f + __expf(-input[i])); // Use fast intrinsic function for exp
    }
}