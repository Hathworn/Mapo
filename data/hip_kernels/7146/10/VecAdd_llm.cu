#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(const int* __restrict__ A, const int* __restrict__ B, int* __restrict__ C, int N) {
    // Compute global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Bounds checking with conditional
    if (i < N) {
        C[i] = A[i] + B[i]; // Perform vector addition
    }
}