#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDot(const int nThreads, const float *m1, const float *m2, float *output, const int m1_rows, const int m1_columns, const int m2_columns) {
    // Use shared memory to reduce global memory accesses
    extern __shared__ float shared_m2[];
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < m1_rows && c < m2_columns) {
        float t_output = 0.0f;
        for (int k = 0; k < m1_columns; ++k) {
            if (threadIdx.y == 0) {
                // Load m2 into shared memory
                shared_m2[threadIdx.x + k * blockDim.x] = m2[k * m2_columns + c];
            }
            __syncthreads(); // Synchronize to ensure shared memory is populated
            t_output += m1[r * m1_columns + k] * shared_m2[threadIdx.x + k * blockDim.x];
            __syncthreads(); // Synchronize threads before next iteration
        }
        output[r * m2_columns + c] = t_output;
    }
}