#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAdd(const int* __restrict__ A, const int* __restrict__ B, int* __restrict__ C, int N) {
    // Use a more efficient method to calculate the global index
    int i = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;

    // Ensure threads do useful work only
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}