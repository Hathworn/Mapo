#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMartixSubstractMatrix(const int nThreads, const float *m1, const float *m2, float *output) {
    // Calculating global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensures we do not access beyond array bounds
    if (i < nThreads) {
        output[i] = m1[i] - m2[i];
    }
}