#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSigmoid_d(const int nThreads, float const *input, float *output) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride loop for better workload distribution
    int stride = blockDim.x * gridDim.x;

    // Loop over all elements using the calculated stride
    for (; i < nThreads; i += stride) {
        // Apply the sigmoid derivative function
        output[i] = input[i] * (1.0f - input[i]);
    }
}