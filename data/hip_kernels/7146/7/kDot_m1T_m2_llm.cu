#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDot_m1T_m2(const int nThreads, const float *m1, const float *m2, float *output, const int m1_rows, const int m1_columns, const int m2_columns ){
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nThreads) {
        // Determine row and column for output
        int r = i / m2_columns;
        int c = i % m2_columns;
        int id_T;
        float t_output = 0.0;

        // Compute dot product for transposed m1 and m2
        for (int k = 0; k < m1_rows; ++k) {
            id_T = k * m1_columns + r;
            t_output += m1[id_T] * m2[k * m2_columns + c];
        }

        // Update output matrix
        output[i] += t_output;
    }
}