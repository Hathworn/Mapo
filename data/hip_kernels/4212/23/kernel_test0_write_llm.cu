#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    // Calculate the starting pointer for the current block
    unsigned int* orig_ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* ptr = orig_ptr;
    
    // Early exit if the start pointer is out of bounds
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }
    
    // Define the end of the current block
    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);
    unsigned int pattern = 1;
    unsigned long mask = 4;
    
    // Set the initial value at the start of the block
    *ptr = pattern;
    
    // Iterate over block memory using calculated mask positions
    while (ptr < block_end) {
        // Calculate new pointer with mask
        ptr = (unsigned int*)(((unsigned long)orig_ptr) | mask);
        if (ptr == orig_ptr) {
            // If it points to the origin again, shift the mask
            mask <<= 1;
            continue;
        }
        if (ptr >= block_end) {
            // Break if the new pointer is out of block bounds
            break;
        }
        
        // Write pattern to the calculated pointer
        *ptr = pattern;
        
        // Update pattern and mask
        pattern <<= 1;
        mask <<= 1;
    }
}