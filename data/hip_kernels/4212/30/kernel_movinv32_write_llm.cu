#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Check if pointer is out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }
    
    // Calculate number of elements to process per thread
    unsigned int threadCount = BLOCKSIZE / sizeof(unsigned int);
    unsigned int tid = threadIdx.x;
    
    // Shared memory allocation for pattern calculation
    __shared__ unsigned int shared_pattern[BLOCKSIZE / sizeof(unsigned int)];
    
    // Offset-based pattern initialization for this thread
    unsigned int pat = (tid == 0) ? pattern : 0;
    unsigned int k = (tid == 0) ? offset : 0;

    // Initialize patterns in shared memory
    for (unsigned int i = tid; i < threadCount; i += blockDim.x) {
        if (tid == 0) {
            shared_pattern[i] = pat;
            k++;
            if (k >= 32) {
                k = 0;
                pat = lb;
            } else {
                pat = (pat << 1) | sval;
            }
        }
    }
    __syncthreads();
    
    // Write patterns from shared memory to global memory
    for (unsigned int i = tid; i < threadCount; i += blockDim.x) {
        ptr[i] = shared_pattern[i];
    }
    
    return;
}