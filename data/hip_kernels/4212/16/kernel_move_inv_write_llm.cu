#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    // Calculate the initial pointer position for this block
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Exit if the pointer is out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }
    
    // Calculate the effective size of the block
    unsigned int elems_per_block = BLOCKSIZE / sizeof(unsigned int);
    
    // Use thread index to parallelize writing within a block
    unsigned int thread_id = threadIdx.x;
    
    // Loop using threads to write pattern, unroll the loop for efficiency
    #pragma unroll
    for (unsigned int i = thread_id; i < elems_per_block; i += blockDim.x) {
        ptr[i] = pattern;
    }
    
    return;
}