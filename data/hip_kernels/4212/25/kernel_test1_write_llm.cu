#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    // Calculate starting index for this block only once
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Check boundary condition before proceeding with loop
    if (ptr >= (unsigned long*) end_ptr) {
        return;
    }

    // Unroll the loop to reduce loop overhead and improve memory access patterns
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned long); i++) {
        ptr[i] = (unsigned long) &ptr[i];
    }
}