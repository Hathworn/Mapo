#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int half_count = BLOCKSIZE / sizeof(unsigned int) / 2;
    unsigned int* ptr_mid = ptr + half_count;

    // Use shared memory for faster access
    __shared__ unsigned int shared_mem[BLOCKSIZE / sizeof(unsigned int)];

    // Load data into shared memory first
    for (unsigned int i = 0; i < half_count; i++) {
        shared_mem[i] = ptr[i];
    }
    
    // Copy from shared memory to destination
    for (unsigned int i = 0; i < half_count - 8; i++) {
        ptr[i + 8] = shared_mem[i];
    }

    // Directly write remaining 8 elements
    for (unsigned int i = 0; i < 8; i++) {
        ptr[i] = shared_mem[half_count - 8 + i];
    }
}