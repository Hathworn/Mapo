#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* end = (unsigned int*) end_ptr;

    // Return early if pointer is out of bounds
    if (ptr >= end) {
        return;
    }

    // Calculate end pointer for this block to avoid boundary pitfalls
    unsigned int* block_end = ptr + BLOCKSIZE / sizeof(unsigned int);
    if (block_end > end) {
        block_end = end;
    }

    // Efficient write using pointer arithmetic and memory coalescing
    for (; ptr < block_end; ++ptr) {
        *ptr = pattern;
    }
}
```
