#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    // Calculate the pointer offset outside the loop to improve efficiency
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* start_ptr = (unsigned int*)_start_ptr;

    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }
    
    // Unroll loop to optimize memory access by reducing loop overhead
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        ptr[i] = start_ptr[i];
    }

    return;
}