#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);

    // Early exit if pointer exceeds end_ptr
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Use shared memory to reduce global memory access
    __shared__ unsigned int shared_block[BLOCKSIZE/sizeof(unsigned int)];

    // Initialize shared memory
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE/sizeof(unsigned int); i += blockDim.x) {
        shared_block[i] = p2; // Set default to p2
    }

    // Synchronize threads
    __syncthreads();

    // Write p1 at specific offsets
    for (unsigned int i = offset + threadIdx.x; i < BLOCKSIZE/sizeof(unsigned int); i += blockDim.x * MOD_SZ) {
        shared_block[i] = p1;
    }

    // Synchronize threads before writing back to global memory
    __syncthreads();

    // Write back to global memory
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE/sizeof(unsigned int); i += blockDim.x) {
        ptr[i] = shared_block[i];
    }
}