#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeLocalMemory(float *output, int size, int repeat)
{
    int gid = threadIdx.x + (blockDim.x * blockIdx.x);
    int tid = threadIdx.x, localSize = blockDim.x, litems = 2048 / localSize;
    int s = tid;
    __shared__ float lbuf[2048];

    // Unroll the loop to optimize for memory access pattern
    for (int j = 0; j < repeat; ++j)
    {
        #pragma unroll 16
        for (int k = 0; k < 16; ++k)
        {
            lbuf[(s + k) & 2047] = gid;
        }
        s = (s + 16) & 2047;
    }

    __syncthreads();

    // Efficiently write data back to output
    for (int j = 0; j < litems; ++j)
    {
        output[gid] = lbuf[tid];
    }
}