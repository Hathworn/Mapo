#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_one_block(volatile unsigned int *frontier, unsigned int frontier_len, 
                                     volatile unsigned int *cost, volatile int *visited, 
                                     unsigned int *edgeArray, unsigned int *edgeArrayAux, 
                                     unsigned int numVertices, unsigned int numEdges, 
                                     volatile unsigned int *frontier_length, unsigned int num_p_per_mp, 
                                     unsigned int w_q_size) {
    // Calculate the thread ID within the block
    unsigned int thread_id = threadIdx.x;

    // Iterate through the frontier
    for (unsigned int i = thread_id; i < frontier_len; i += blockDim.x) {
        unsigned int vertex = frontier[i];

        // Only proceed if the vertex has not been visited
        if (!visited[vertex]) {
            visited[vertex] = 1;  // Mark as visited

            // Examine the edges of the current vertex
            unsigned int start_edge = edgeArray[vertex];
            unsigned int end_edge = edgeArray[vertex + 1];

            for (unsigned int j = start_edge; j < end_edge; j++) {
                unsigned int neighbor = edgeArrayAux[j];

                // Atomic operation to avoid race conditions
                if (atomicMin(&cost[neighbor], cost[vertex] + 1) > cost[vertex] + 1) {
                    // Add neighbor to the frontier
                    unsigned int index = atomicAdd(frontier_length, 1);

                    if (index < w_q_size) {  // Ensure the frontier size does not exceed the max
                        frontier[index] = neighbor;
                    }
                }
            }
        }
    }
}