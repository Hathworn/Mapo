#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeGlobalMemoryUnit(float *output, int size, int repeat) {
    int gid = threadIdx.x + (blockDim.x * blockIdx.x), j;
    int s = gid * 512;
    
    // Unrolling loop for coalesced global memory access
    for (j = 0; j < repeat; ++j) {
        #pragma unroll
        for (int offset = 0; offset < 16; ++offset) {
            output[(s + offset) & (size - 1)] = gid;
        }
        s = (s + 16) & (size - 1);
    }
}