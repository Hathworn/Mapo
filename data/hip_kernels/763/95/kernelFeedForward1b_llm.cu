#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFeedForward1b(float *zs, int bound, float *weights, int w_off, float *activations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize output value for each thread
    float sum = 0.0f;
    
    // Use shared memory to improve memory access patterns (for inputs, if applicable) 
    extern __shared__ float shared_activations[];
    for (int i = threadIdx.x; i < bound; i += blockDim.x) {
        shared_activations[i] = activations[blockIdx.x * bound + i];
    }
    __syncthreads();

    // Compute the dot product
    for (int i = 0; i < bound; i++) {
        sum += weights[w_off + (threadIdx.x * bound) + i] * shared_activations[i];
    }
    
    // Store the result
    zs[idx] = sum;
}