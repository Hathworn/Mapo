#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void executeThirdLayer(float *Layer3_Neurons_GPU, float *Layer3_Weights_GPU, float *Layer4_Neurons_GPU)
{
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int weightBegin = blockID * 1251;
    
    float result = 0.0f;

    // Use shared memory to load weights if necessary, for improved memory access
    __shared__ float sharedWeights[1251]; 

    if (threadID == 0) {
        sharedWeights[0] = Layer3_Weights_GPU[weightBegin]; // Load bias
    }

    // Load remaining weights into shared memory
    for (int i = threadID; i < 1250; i += blockDim.x) {
        sharedWeights[i + 1] = Layer3_Weights_GPU[weightBegin + 1 + i];
    }
    __syncthreads(); // Ensure all weights are loaded before proceeding

    // Parallel accumulate the result
    float localResult = 0.0f;
    for (int i = threadID; i < 1250; i += blockDim.x) {
        localResult += Layer3_Neurons_GPU[i + (1250 * blockIdx.y)] * sharedWeights[i + 1];
    }

    // Perform reduction of results from all threads
    atomicAdd(&result, localResult);
    __syncthreads(); // Ensure all partial results are accumulated

    if (threadID == 0) {
        result += sharedWeights[0]; // Add bias
        result = 1.7159f * tanhf(0.66666667f * result);
        Layer4_Neurons_GPU[blockID + (100 * blockIdx.y)] = result;
    }
}