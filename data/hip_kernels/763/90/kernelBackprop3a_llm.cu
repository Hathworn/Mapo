#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelBackprop3a(float *delta_nabla_b, int b_off, int bound, int b_off_old, float *weights, int w_off_old) {

    // Cache the delta_nabla_b value in a register for faster access
    float delta_nb = delta_nabla_b[b_off + threadIdx.x];
    
    // Initialize the register
    delta_nb = 0.0;

    for (int j = 0; j < bound; j++) {
        // Calculate the weights index once outside the for loop to reduce redundant computation
        int weight_idx = w_off_old + (j * blockDim.x) + threadIdx.x;
        
        // Accumulate directly into the register
        delta_nb += delta_nabla_b[b_off_old + j] * weights[weight_idx];
    }

    // Write the computed value back to global memory
    delta_nabla_b[b_off + threadIdx.x] = delta_nb;
}