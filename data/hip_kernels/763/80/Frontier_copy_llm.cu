#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function
__global__ void Frontier_copy(unsigned int *frontier, unsigned int *frontier2, unsigned int *frontier_length) {
    // Calculate global thread ID for current execution
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Only threads within range execute this copy
    if (tid < *frontier_length) {
        // Copy data from frontier2 to frontier
        frontier[tid] = frontier2[tid];
    }
}