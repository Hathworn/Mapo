#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFeedForward1(float *zs, int bound2, float *weights, int w_off, float *activations1) {
    // Calculate unique index for each thread
    int idx = threadIdx.x;

    // Initialize output
    float sum = 0.0;

    // Use loop unrolling to optimize accumulation
    int i;
    for (i = 0; i < bound2 - 3; i += 4) {
        sum += weights[w_off + (idx * bound2) + i] * activations1[i];
        sum += weights[w_off + (idx * bound2) + i + 1] * activations1[i + 1];
        sum += weights[w_off + (idx * bound2) + i + 2] * activations1[i + 2];
        sum += weights[w_off + (idx * bound2) + i + 3] * activations1[i + 3];
    }

    // Handle remaining elements if bound2 is not divisible by 4
    for (; i < bound2; i++) {
        sum += weights[w_off + (idx * bound2) + i] * activations1[i];
    }

    // Write the result back to global memory
    zs[idx] = sum;
}