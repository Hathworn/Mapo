#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelUpdateBiases(float *nabla_b, float *biases, float eta, float mini_batch_size) {
    // Compute rate once and use it for all threads to improve efficiency
    float rate = eta / mini_batch_size;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    biases[index] -= rate * nabla_b[index]; // Utilize blockIdx for accessing elements beyond blockDim.x
}