#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFeedForward2(float *zs, float *biases, int b_off, float *activations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global index

    if (idx < b_off) {  // Bounds checking
        zs[idx] += biases[b_off + idx];  // Update zs using biases
        activations[idx] = 1.0f / (1.0f + expf(-zs[idx]));  // Compute activation
    }
}