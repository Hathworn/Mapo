#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_SM_block(volatile unsigned int *frontier, volatile unsigned int *frontier2, unsigned int frontier_len, 
                                    volatile unsigned int *cost, volatile int *visited, unsigned int *edgeArray, 
                                    unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, 
                                    volatile unsigned int *frontier_length, unsigned int num_p_per_mp, unsigned int w_q_size) 
{
    // Calculate global thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check thread ID within frontier length for bounds safety
    if (tid < frontier_len) {
        unsigned int vertex = frontier[tid];

        // Iterate over adjacent vertices
        for (unsigned int i = edgeArrayAux[vertex]; i < edgeArrayAux[vertex + 1]; ++i) {
            unsigned int neighbor = edgeArray[i];

            // Atomic operation ensures correct updating of shared data (visited)
            if (atomicExch(&visited[neighbor], 1) == 0) {
                // Update with appropriate BFS tree cost
                cost[neighbor] = cost[vertex] + 1;

                // Determine position in frontier2, increment frontier_length atomically
                unsigned int pos = atomicAdd((unsigned int*)frontier_length, 1);

                // Assign vertex to frontier2 at calculated position
                frontier2[pos] = neighbor;
            }
        }
    }
}