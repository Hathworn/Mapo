#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readLocalMemory(const float *data, float *output, int size, int repeat)
{
    int gid = threadIdx.x + (blockDim.x * blockIdx.x);
    float sum = 0;
    int tid = threadIdx.x, localSize = blockDim.x, grpid = blockIdx.x;
    int litems = 2048 / localSize, goffset = localSize * grpid + tid * litems;
    int s = tid;
    __shared__ float lbuf[2048];

    // Use coalesced memory access
    for (int j = 0; j < litems && (j + goffset) < size ; ++j)
    {
        lbuf[tid * litems + j] = data[goffset + j];
    }
    __syncthreads();

    for (int j = 0; j < repeat; ++j)
    {
        // Unroll loop for better performance
        #pragma unroll
        for (int i = 0; i < 16; ++i)
        {
            sum += lbuf[(s + i) & 2047];
        }
        s = (s + 16) & 2047;
    }
    output[gid] = sum;
}