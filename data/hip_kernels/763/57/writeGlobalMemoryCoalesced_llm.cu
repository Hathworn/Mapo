#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeGlobalMemoryCoalesced(float *output, int size, int repeat)
{
    int gid = threadIdx.x + (blockDim.x * blockIdx.x);
    int s = gid;
    int offset[] = {0, 32768, 65536, 98304, 131072, 163840, 196608, 229376, 
                    262144, 294912, 327680, 360448, 393216, 425984, 458752, 491520};

    // Loop unrolling for better performance
    for (int j=0; j<repeat; ++j)
    {
        #pragma unroll
        for (int i=0; i<16; ++i)
        {
            output[(s + offset[i]) & (size-1)] = gid;
        }
        s = (s + 524288) & (size-1);
    }
}
```
