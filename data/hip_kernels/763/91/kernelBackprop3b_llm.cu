#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelBackprop3b(float *delta_nabla_b, int b_off, float *zs) {
    // Use shared memory to reduce redundant computations
    __shared__ float shared_zs[256]; // Assuming max 256 threads per block

    int idx = threadIdx.x;
    shared_zs[idx] = zs[idx];
    __syncthreads();

    // Calculate sigmoid only once
    float sigmoid = 1.0f / (1.0f + expf(-shared_zs[idx]));
    delta_nabla_b[b_off + idx] *= sigmoid * (1.0f - sigmoid);
}