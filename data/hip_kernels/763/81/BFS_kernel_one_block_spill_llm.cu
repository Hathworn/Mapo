#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_one_block_spill(volatile unsigned int *frontier, unsigned int frontier_len, volatile unsigned int *cost, volatile int *visited, unsigned int *edgeArray, unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, volatile unsigned int *frontier_length, const unsigned int max_mem) {
    extern __shared__ unsigned int shared_memory[]; // Use shared memory

    unsigned int *local_frontier = shared_memory;
    unsigned int tid = threadIdx.x;

    for (int i = tid; i < frontier_len; i += blockDim.x) {
        unsigned int node = frontier[i];
        
        unsigned int start = edgeArray[node];
        unsigned int end = edgeArray[node + 1];

        for (unsigned int edge = start; edge < end; ++edge) {
            unsigned int neighbor = edgeArrayAux[edge];
            
            if (!visited[neighbor]) {
                visited[neighbor] = 1;
                cost[neighbor] = cost[node] + 1;
                unsigned int index = atomicAdd(frontier_length, 1);
                
                if (index < max_mem) {
                    local_frontier[index] = neighbor; // Add to local frontier
                }
            }
        }
    }

    __syncthreads(); // Synchronize threads before writing back to global memory

    for (int i = tid; i < *frontier_length; i += blockDim.x) {
        if (i < max_mem) {
            frontier[i] = local_frontier[i]; // Write back to global memory
        }
    }
}