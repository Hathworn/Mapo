#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void kernelBackprop1(float *delta_nabla_w, int w_off, float *activations, float *delta_nabla_b, int b_off) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global index
    delta_nabla_w[w_off + idx] = activations[threadIdx.x] * delta_nabla_b[b_off + blockIdx.x];
}