#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void executeFourthLayer(float *Layer4_Neurons_GPU, float *Layer4_Weights_GPU, float *Layer5_Neurons_GPU)
{
    int blockID = blockIdx.x;
    int weightBegin = blockID * 101;

    float result = 0;

    // Access first weight
    result += __ldg(&Layer4_Weights_GPU[weightBegin]);
    ++weightBegin;

    // Loop unrolling for optimization
    #pragma unroll 4
    for (int i = 0; i < 100; ++i)
    {
        result += __ldg(&Layer4_Neurons_GPU[i + (100 * blockIdx.y)]) * __ldg(&Layer4_Weights_GPU[weightBegin + i]);
    }

    result = (1.7159f * tanhf(0.66666667f * result));

    Layer5_Neurons_GPU[blockID + (10 * blockIdx.y)] = result;
}