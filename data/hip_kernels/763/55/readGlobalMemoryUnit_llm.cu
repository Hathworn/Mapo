#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readGlobalMemoryUnit(float *data, float *output, int size, int repeat)
{
    int gid = threadIdx.x + (blockDim.x * blockIdx.x);
    float sum = 0;
    int s = gid * 512;
    // Unroll loop and simplify addressing
    for (int j = 0; j < repeat; ++j) {
        #pragma unroll
        for (int k = 0; k < 16; ++k) {
            sum += data[(s + k) & (size - 1)];
        }
        s = (s + 16) & (size - 1);
    }
    output[gid] = sum;
}