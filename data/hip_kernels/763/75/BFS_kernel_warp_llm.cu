#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_warp( unsigned int *levels, unsigned int *edgeArray, unsigned int *edgeArrayAux, int W_SZ, int CHUNK_SZ, unsigned int numVertices, int curr, int *flag) {
    // Calculate the global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use warp-synchronous programming to improve access pattern and reduce divergence
    int warpId = tid / W_SZ;  // Determine the warp ID
    int laneId = tid % W_SZ;  // Determine the lane ID within the warp
    
    // Process each chunk by iterating in steps of warp size
    for (int vertexIdx = warpId * CHUNK_SZ; vertexIdx < numVertices; vertexIdx += gridDim.x * CHUNK_SZ) {
        if (vertexIdx + laneId < numVertices) {
            // Read and process each edge in the chunk
            unsigned int vertex = vertexIdx + laneId;
            if (levels[vertex] == curr) {
                for (unsigned int edgeIdx = edgeArrayAux[vertex]; edgeIdx < edgeArrayAux[vertex + 1]; edgeIdx++) {
                    unsigned int neighbor = edgeArray[edgeIdx];

                    // Atomically update the level for each neighbor
                    if (atomicCAS(&levels[neighbor], UINT_MAX, curr + 1) == UINT_MAX) {
                        *flag = 1;  // Set the flag if any new node is discovered
                    }
                }
            }
        }
    }
}