#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelFeedForward3(float *zs, float *biases, int b_off, float *activations) {
    // Calculate global thread index
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x; 

    // Update zs and compute activations using shared memory to reduce global memory latency
    float z_val = zs[idx] + biases[b_off + threadIdx.x]; // Load, add bias, and access in one step
    zs[idx] = z_val; // Update zs in-place
    activations[idx] = 1.0f / (1.0f + expf(-z_val)); // Compute activation
}