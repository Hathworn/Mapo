#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_SM_block_spill(volatile unsigned int *frontier, volatile unsigned int *frontier2, unsigned int frontier_len, volatile unsigned int *cost, 
    volatile int *visited, unsigned int *edgeArray, unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, 
    volatile unsigned int *frontier_length, const unsigned int max_mem) {
    
    // Calculate the global thread index
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= frontier_len) return; // Ensure we only process valid frontier elements

    // Load frontier element into a register for faster access
    unsigned int currentVertex = frontier[tid];

    // Process edges of the current vertex
    unsigned int startEdge = edgeArrayAux[currentVertex];
    unsigned int endEdge = edgeArrayAux[currentVertex + 1];

    // Iterate over neighbors
    for (unsigned int edge = startEdge; edge < endEdge; ++edge) {
        unsigned int neighbor = edgeArray[edge];
        
        // Check if the neighbor has been visited
        if (visited[neighbor] == 0) {
            // Mark neighbor as visited
            visited[neighbor] = 1;

            // Update cost for the neighbor
            cost[neighbor] = cost[currentVertex] + 1;

            // Add neighbor to the new frontier if within memory limits
            unsigned int index = atomicAdd((unsigned int *)frontier_length, 1);
            if (index < max_mem) {
                frontier2[index] = neighbor;
            }
        }
    }
}