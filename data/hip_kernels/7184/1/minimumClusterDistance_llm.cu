#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DIMENSIONS 2
#define GPU_DEVICE_ZERO 0

__global__ void minimumClusterDistance(int threads, double *pointToClusterDistance, int *minimumPointToCluster, int pointsCounter, int clusterCounter)
{
    // Calculate global thread index
    int pointIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Only proceed if within bounds
    if (pointIndex < pointsCounter) 
    {
        int smallestIndex = 0;
        double min = pointToClusterDistance[pointIndex]; // Initialize min with first element
        
        // Loop through each cluster
        for (int index = 1; index < clusterCounter; ++index) 
        {
            double temp = pointToClusterDistance[pointIndex + index * pointsCounter];
            // Update minimum if a smaller distance is found
            if (temp < min) 
            {
                min = temp;
                smallestIndex = index; // Save new index of minimum distance
            }
        }
        
        minimumPointToCluster[pointIndex] = smallestIndex; // Assign nearest cluster index
    }
}