#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"


#define DIMENSIONS 2
#define GPU_DEVICE_ZERO 0



__global__ void distanceArrCalc(int pointsCounter, int threads, double *distanceFromPointToCluster, double *pointsInGpu, double *clustersInGpu)
{
    // Use shared memory to optimize data access
    __shared__ double sharedClusters[DIMENSIONS * blockDim.y];

    double distanceX = 0;
    double distanceY = 0;
    
    // Calculate and load clusters data into shared memory
    if (threadIdx.x < blockDim.y) {
        int index = threadIdx.x * DIMENSIONS;
        sharedClusters[index] = clustersInGpu[index];
        sharedClusters[index + 1] = clustersInGpu[index + 1];
    }
    __syncthreads();

    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (globalThreadId < pointsCounter) {
        int offsetPointIndex = globalThreadId * DIMENSIONS;
        int offsetClusterIndexForPoint = threadIdx.y * DIMENSIONS;

        // Load points data
        double pointX = pointsInGpu[offsetPointIndex];
        double pointY = pointsInGpu[offsetPointIndex + 1];

        // Load clusters data from shared memory
        double clusterX = sharedClusters[offsetClusterIndexForPoint];
        double clusterY = sharedClusters[offsetClusterIndexForPoint + 1];

        // Calculate squared distances
        distanceX = (pointX - clusterX) * (pointX - clusterX);
        distanceY = (pointY - clusterY) * (pointY - clusterY);

        // Compute total distance and store result
        double totalDistance = sqrt(distanceX + distanceY);
        int currentPointIndexY = pointsCounter * threadIdx.y;
        int pointIndex = currentPointIndexY + globalThreadId;
        distanceFromPointToCluster[pointIndex] = totalDistance;
    }
}