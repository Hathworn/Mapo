#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DIMENSIONS 2
#define GPU_DEVICE_ZERO 0

__global__ void pointToThreadMove(int pointsCounter, int threadsInsideBlock, double dt, double *pointsInGpu, double *speedArrayInGpu)
{
    // Calculate global thread ID once
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalThreadId < pointsCounter)
    {
        // Use a loop unrolling technique to enhance performance
        #pragma unroll
        for (int indexInArray = 0; indexInArray < DIMENSIONS; ++indexInArray)
        {
            // Precompute the index to reduce redundant calculation
            int currentGpuPoint = globalThreadId * DIMENSIONS + indexInArray;
            pointsInGpu[currentGpuPoint] += speedArrayInGpu[currentGpuPoint] * dt;
        }
    }
}