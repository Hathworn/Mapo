#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mat_sub(float *a, float *b, float *c, int rows, int columns)
{
    // Calculate 1D index from 2D grid
    int index = blockIdx.y * blockDim.x * blockDim.y + blockIdx.x * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * gridDim.y * blockDim.x * blockDim.y;
    
    // Use stride loop to process all elements
    for (int k = index; k < rows * columns; k += total_threads) {
        c[k] = a[k] - b[k];
    }
}