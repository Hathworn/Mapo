#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mat_scale(float alpha, float *a, float *c, int rows, int columns, int depth)
{
    // Calculate flattened 3D index for linear memory access
    int idx = blockDim.y * blockIdx.y + threadIdx.y;
    int idy = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (idx < rows && idy < columns) {
        int baseIndex = (idx * columns + idy) * depth;
        for (int k = 0; k < depth; k++) {
            // Perform scaling computation
            c[baseIndex + k] = alpha * a[baseIndex + k];
        }
    }
}