#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv(float *t, float *tk, float *out, int t_rows, int t_columns, int n_channels, int k_rows, int k_columns, int n_kernels)
{
    const int i_out = blockDim.y * blockIdx.y + threadIdx.y;
    const int j_out = blockDim.x * blockIdx.x + threadIdx.x;

    int i0 = i_out - k_rows / 2;
    int j0 = j_out - k_columns / 2;

    if (i_out < t_rows && j_out < t_columns) {
        for (int k = 0; k < n_kernels; k++) {
            float convolution = 0.0f;

            // Optimize loops using shared memory (assuming it's properly sized)
            __shared__ float shared_t[/*appropriate size*/];
            __shared__ float shared_tk[/*appropriate size*/];

            // Load data into shared memory
            shared_t[threadIdx.y * blockDim.x + threadIdx.x] = t[(i_out * t_columns + j_out) * n_channels + threadIdx.y];
            shared_tk[threadIdx.y * blockDim.x + threadIdx.x] = tk[threadIdx.y * k_columns + threadIdx.x * n_kernels + k];

            __syncthreads(); // Synchronize threads

            for (int m = 0; m < k_rows; m++) {
                for (int n = 0; n < k_columns; n++) {
                    for (int c = 0; c < n_channels; c++) {
                        int i_index = i0 + m;
                        int j_index = j0 + n;

                        if (i_index >= 0 && i_index < t_rows && j_index >= 0 && j_index < t_columns) {
                            convolution += shared_t[(i_index * t_columns + j_index) * n_channels + c] * shared_tk[(m * k_columns + n) * n_kernels + k];
                        }
                    }
                }
            }

            out[(i_out * t_columns + j_out) * n_kernels + k] = convolution;
        }
    }
}