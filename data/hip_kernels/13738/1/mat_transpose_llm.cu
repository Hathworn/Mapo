#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mat_transpose(float *a, float *out, int size_x, int size_y)
{
    __shared__ float tile[32][33]; // Use shared memory to improve access patterns

    const int tile_x = threadIdx.x;
    const int tile_y = threadIdx.y;
    const int x = blockIdx.x * blockDim.x + tile_x;
    const int y = blockIdx.y * blockDim.y + tile_y;

    if (x < size_x && y < size_y) {
        tile[tile_y][tile_x] = a[y * size_y + x]; // Load data into shared memory
    }
    __syncthreads();

    const int transposed_x = blockIdx.y * blockDim.y + tile_x;
    const int transposed_y = blockIdx.x * blockDim.x + tile_y;

    if (transposed_x < size_y && transposed_y < size_x) {
        out[transposed_x * size_x + transposed_y] = tile[tile_x][tile_y]; // Write transposed data
    }
}