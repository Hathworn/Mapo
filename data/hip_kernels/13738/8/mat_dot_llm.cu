#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mat_dot(float *a, float *b, float *c, int a_rows, int a_columns, int b_rows, int b_columns)
{
    const int i = blockDim.y * blockIdx.y + threadIdx.y;
    const int j = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < a_rows && j < b_columns)
    {
        float c_at_ij = 0;
        // Use shared memory to reduce global memory accesses
        __shared__ float sharedA[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float sharedB[BLOCK_SIZE][BLOCK_SIZE];

        for (int tile = 0; tile < (a_columns + BLOCK_SIZE - 1) / BLOCK_SIZE; ++tile) 
        {
            if (tile * BLOCK_SIZE + threadIdx.x < a_columns && i < a_rows)
                sharedA[threadIdx.y][threadIdx.x] = a[i * a_columns + tile * BLOCK_SIZE + threadIdx.x];
            else
                sharedA[threadIdx.y][threadIdx.x] = 0.0f;

            if (tile * BLOCK_SIZE + threadIdx.y < b_rows && j < b_columns) 
                sharedB[threadIdx.y][threadIdx.x] = b[(tile * BLOCK_SIZE + threadIdx.y) * b_columns + j];
            else
                sharedB[threadIdx.y][threadIdx.x] = 0.0f;

            __syncthreads();

            for (int k = 0; k < BLOCK_SIZE; ++k)
                c_at_ij += sharedA[threadIdx.y][k] * sharedB[k][threadIdx.x];

            __syncthreads();
        }

        c[i * b_columns + j] = c_at_ij;
    }
}