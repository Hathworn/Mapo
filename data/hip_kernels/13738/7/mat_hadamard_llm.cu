#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mat_hadamard(float *a, float *b, float *c, int rows, int columns)
{
    int k = blockDim.x * blockIdx.x + threadIdx.x; // Flatten the 2D grid and block into 1D index

    if (k < rows * columns) // Use the 1D index to check bounds
    {
        c[k] = a[k] * b[k]; // Perform element-wise multiplication
    }
}