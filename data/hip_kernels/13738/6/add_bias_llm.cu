#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_bias(float *a, float *bias, float *out, int size_x, int size_y, int size_z)
{
    // Calculate global thread indices
    const int i = blockIdx.y * blockDim.y + threadIdx.y;
    const int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds
    if (i < size_x && j < size_y)
    {
        // Precompute the starting index
        int k = (i * size_y + j) * size_z;

        // Use a pointer for iteration to reduce indexing computation
        float* out_ptr = &out[k];
        float* a_ptr = &a[k];

        // Use loop unrolling for better performance (assuming size_z is known)
        for (int c = 0; c < size_z; ++c)
        {
            out_ptr[c] = a_ptr[c] + bias[c];
        }
    }
}