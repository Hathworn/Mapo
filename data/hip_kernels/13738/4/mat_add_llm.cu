#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mat_add(float *a, float *b, float *c, int limit)
{
    // Calculate global thread ID
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Load data from global memory to registers for optimal access
    if (i < limit)
    {
        float a_val = a[i];
        float b_val = b[i];

        // Compute addition and store result
        c[i] = a_val + b_val;
    }
}