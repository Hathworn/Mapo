#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void t_sum(float *a, float *out, int n_elements)
{
    // Calculate the global thread index
    const int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Initialize out[0] only once before any atomic operations
    if (i == 0) out[0] = 0;
    __syncthreads();

    // Perform atomic addition only if the index is within bounds
    if (i < n_elements)
    {
        atomicAdd(out, a[i]);
    }
}