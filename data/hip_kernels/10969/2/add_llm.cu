#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Use shared memory to reduce global memory accesses
    __shared__ int s_a, s_b;
    
    // Load inputs from global memory to shared memory
    s_a = a[blockIdx.x];
    s_b = b[blockIdx.x];
    
    // Ensure all loads to shared memory are completed
    __syncthreads();
    
    // Compute and store the result back to global memory
    c[blockIdx.x] = s_a + s_b;
}