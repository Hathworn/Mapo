#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Utilize block and thread indices to support parallel execution
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we don't access out of bounds
    if (idx == 0)
    {
        c[idx] = a[idx] + b[idx];
    }
}