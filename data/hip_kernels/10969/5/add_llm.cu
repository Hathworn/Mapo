#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void add(int *a, int *b, int *c)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Compute global index
    c[idx] = a[idx] + b[idx]; // Use global index to access array elements
}