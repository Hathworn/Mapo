#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Nmber of blocks
#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c)
{
    // Calculate unique global index for each thread
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure index does not exceed the array bounds
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}