#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sharedMatmul(float *a, float *b, float *c, int n) {
    __shared__ float A_tile[32][32];
    __shared__ float B_tile[32][32];
    int width = gridDim.x * blockDim.x;

    float acc = 0;
    int i = blockIdx.x * 32 + threadIdx.x;
    int j = blockIdx.y * 32 + threadIdx.y;

    // Loop unrolling to improve performance
    for (int tileIdx = 0; tileIdx < gridDim.x; tileIdx++) {
        A_tile[threadIdx.y][threadIdx.x] = a[j * width + tileIdx * 32 + threadIdx.x];
        B_tile[threadIdx.y][threadIdx.x] = b[(tileIdx * 32 + threadIdx.y) * width + i];
        
        __syncthreads();

        // Use registers to accumulate result to reduce memory access latency
        #pragma unroll
        for (int k = 0; k < 32; k++) {
            acc += A_tile[threadIdx.y][k] * B_tile[k][threadIdx.x];
        }

        __syncthreads();
    }

    c[j * width + i] = acc;
}