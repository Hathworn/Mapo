#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void optimalTransposeKernel(const float *input, float *output, int n)
{
    // Use shared memory for efficient block-wise transpose
    __shared__ float tile[64][65];
    // Calculate global x and y indices
    int global_x = blockIdx.x * 64 + threadIdx.x;
    int global_y = blockIdx.y * 64 + threadIdx.y;
    // Constants for maximum width and height based on grid dimensions
    const int max_width = gridDim.x * 64;
    const int max_height = gridDim.y * 64;
    
    // Load data into shared memory in a coalesced fashion
    if (global_x < max_width && global_y < max_height) {
        tile[threadIdx.y][threadIdx.x] = input[global_y * max_width + global_x];
        tile[threadIdx.y + 16][threadIdx.x] = input[(global_y + 16) * max_width + global_x];
        tile[threadIdx.y + 32][threadIdx.x] = input[(global_y + 32) * max_width + global_x];
        tile[threadIdx.y + 48][threadIdx.x] = input[(global_y + 48) * max_width + global_x];
    }
    __syncthreads();

    // Calculate transposed x and y indices for output
    int transposed_x = blockIdx.y * 64 + threadIdx.x;
    int transposed_y = blockIdx.x * 64 + threadIdx.y;
    
    // Store transposed data from shared memory back to global memory
    if (transposed_y < max_width && transposed_x < max_height) {
        output[transposed_y * max_height + transposed_x] = tile[threadIdx.x][threadIdx.y];
        output[(transposed_y + 16) * max_height + transposed_x] = tile[threadIdx.x][threadIdx.y + 16];
        output[(transposed_y + 32) * max_height + transposed_x] = tile[threadIdx.x][threadIdx.y + 32];
        output[(transposed_y + 48) * max_height + transposed_x] = tile[threadIdx.x][threadIdx.y + 48];
    }
}