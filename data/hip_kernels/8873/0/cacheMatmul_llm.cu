#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cacheMatmul(float *a, float *b, float *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < n && j < n) { // Ensure within bounds
        float acc = 0.0f; // Initialize accumulator
        for (int k = 0; k < n; ++k) { // Optimize by iterating over full range
            acc += a[i * n + k] * b[k * n + j]; // Perform multiplication and accumulation
        }
        c[i * n + j] = acc; // Write result back to c
    }
}