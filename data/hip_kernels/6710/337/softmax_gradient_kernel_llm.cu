#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // Efficient reduction using single pass per block
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Parallel reduction within shared memory
    #pragma unroll
    for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
        if (idx < offset) {
            reduction_buffer[idx] += reduction_buffer[idx + offset];
        }
        __syncthreads();
    }
    
    // Broadcast result and compute gradients
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}