#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Use fast conditional operator instead of branching logic
        int numMax = 0;
        #pragma unroll  // Unroll loop for better performance
        for (int i = 0; i < numOut; i++) {
            numMax += __float2int_rd(probs[i * numCases + tx] == maxp); // Use fast integer conversion
        }
        correctProbs[tx] = (labelp == maxp) ? (1.0f / float(numMax)) : 0.0f;  // Replace branching with ternary operator
    }
}