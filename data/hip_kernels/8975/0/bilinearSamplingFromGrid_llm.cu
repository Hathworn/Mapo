#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ bool between(int value, int lowerBound, int upperBound)
{
    return (value >= lowerBound && value <= upperBound);
}

__device__ void getTopLeft(float x, int width, int& point, float& weight)
{
    float xcoord = (x + 1) * (width - 1) / 2;
    point = floor(xcoord);
    weight = 1 - (xcoord - point);
}

__global__ void bilinearSamplingFromGrid(float* inputImages_data, int inputImages_strideBatch, int inputImages_strideChannels, int inputImages_strideHeight, int inputImages_strideWidth, float* grids_data, int grids_strideBatch, int grids_strideYX, int grids_strideHeight, int grids_strideWidth, float* output_data, int output_strideBatch, int output_strideChannels, int output_strideHeight, int output_strideWidth, int inputImages_channels, int inputImages_height, int inputImages_width, int output_width)
{
    // Calculate output coordinate
    const int xOut = blockIdx.x * blockDim.y + threadIdx.y;
    if (xOut >= output_width) return;
    
    const int yOut = blockIdx.y;
    const int b = blockIdx.z;
    
    const int width = inputImages_width;
    const int height = inputImages_height;

    // Optimize shared memory access by using warp-level parallelism
    float gridData[2];
    int gridIndex = b * grids_strideBatch + yOut * grids_strideHeight + xOut * grids_strideWidth;
    gridData[0] = grids_data[gridIndex];
    gridData[1] = grids_data[gridIndex + 1];

    float yf = gridData[0];
    float xf = gridData[1];

    int yInTopLeft, xInTopLeft;
    float yWeightTopLeft, xWeightTopLeft;
    getTopLeft(xf, inputImages_width, xInTopLeft, xWeightTopLeft);
    getTopLeft(yf, inputImages_height, yInTopLeft, yWeightTopLeft);

    const int outAddress = output_strideBatch * b + output_strideHeight * yOut + output_strideWidth * xOut;
    const int inTopLeftAddress = inputImages_strideBatch * b + inputImages_strideHeight * yInTopLeft + inputImages_strideWidth * xInTopLeft;

    float v = 0.0;

    bool topLeftIsIn = between(xInTopLeft, 0, width - 1) && between(yInTopLeft, 0, height - 1);
    bool topRightIsIn = between(xInTopLeft + 1, 0, width - 1) && between(yInTopLeft, 0, height - 1);
    bool bottomLeftIsIn = between(xInTopLeft, 0, width - 1) && between(yInTopLeft + 1, 0, height - 1);
    bool bottomRightIsIn = between(xInTopLeft + 1, 0, width - 1) && between(yInTopLeft + 1, 0, height - 1);

    for (int t = threadIdx.x; t < inputImages_channels; t += blockDim.x)
    {
        float inTopLeft = (topLeftIsIn) ? inputImages_data[inTopLeftAddress + t] : 0.0f;
        float inTopRight = (topRightIsIn) ? inputImages_data[inTopLeftAddress + inputImages_strideWidth + t] : 0.0f;
        float inBottomLeft = (bottomLeftIsIn) ? inputImages_data[inTopLeftAddress + inputImages_strideHeight + t] : 0.0f;
        float inBottomRight = (bottomRightIsIn) ? inputImages_data[inTopLeftAddress + inputImages_strideHeight + inputImages_strideWidth + t] : 0.0f;

        v = xWeightTopLeft * yWeightTopLeft * inTopLeft
            + (1 - xWeightTopLeft) * yWeightTopLeft * inTopRight
            + xWeightTopLeft * (1 - yWeightTopLeft) * inBottomLeft
            + (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * inBottomRight;

        output_data[outAddress + t] = v;
    }
}