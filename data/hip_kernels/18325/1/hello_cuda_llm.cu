#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_cuda()
{
    // Check if thread is the first thread in the first block
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        // Print message from one thread only to reduce overhead
        printf("Hello Cuda world\n");
    }
}