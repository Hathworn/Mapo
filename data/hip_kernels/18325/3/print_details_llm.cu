#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_details()
{
    // Use a single printf statement for better readability and efficiency
    printf(
        "blockIdx: (%d, %d, %d), blockDim: (%d, %d, %d), gridDim: (%d, %d, %d)\n",
        blockIdx.x, blockIdx.y, blockIdx.z,
        blockDim.x, blockDim.y, blockDim.z,
        gridDim.x, gridDim.y, gridDim.z
    );
}