#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global index for accessing arrays
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread computes one element, assuming array size matches grid size
    c[i] = a[i] + b[i];
}