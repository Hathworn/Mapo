#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_threadIds()
{
    // Use single printf to reduce overhead 
    printf("Thread ID - x: %d, y: %d, z: %d\n", threadIdx.x, threadIdx.y, threadIdx.z);
}