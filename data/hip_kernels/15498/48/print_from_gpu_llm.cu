#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_from_gpu(void) {
    // Use registers to reduce memory access and improve performance
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    printf("Hello World! from thread [%d,%d] From device\n", tx, bx);
}