#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sgemm_gpu_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Calculate column and row indices for threads
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure that the thread is within matrix bounds
    if (row < N && col < M) {
        float sum = 0.f;

        // Use shared memory for partial matrix tiles to optimize memory access
        __shared__ float sharedA[TILE_DIM][TILE_DIM];
        __shared__ float sharedB[TILE_DIM][TILE_DIM];

        // Iterate over tiles
        for (int t = 0; t < (K + TILE_DIM - 1) / TILE_DIM; ++t) {
            // Load data into shared memory
            if (t * TILE_DIM + threadIdx.x < K && row < N)
                sharedA[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_DIM + threadIdx.x];
            else
                sharedA[threadIdx.y][threadIdx.x] = 0.0f;

            if (t * TILE_DIM + threadIdx.y < K && col < M)
                sharedB[threadIdx.y][threadIdx.x] = B[(t * TILE_DIM + threadIdx.y) * M + col];
            else
                sharedB[threadIdx.y][threadIdx.x] = 0.0f;

            // Synchronize threads to ensure full tile is loaded
            __syncthreads();

            // Compute partial product for the tile
            for (int i = 0; i < TILE_DIM; ++i)
                sum += sharedA[threadIdx.y][i] * sharedB[i][threadIdx.x];

            // Synchronize threads before loading new tile
            __syncthreads();
        }

        // Update output matrix
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}
```
