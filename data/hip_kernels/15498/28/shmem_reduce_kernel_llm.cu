#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shmem_reduce_kernel(float * d_out, const float * d_in)
{
    // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // Optimized: Prefetch data to shared memory only if within boundaries
    sdata[tid] = (myId < gridDim.x * blockDim.x) ? d_in[myId] : 0.0f;
    __syncthreads();

    // Do reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        // Unroll loop for efficiency
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Only thread 0 writes result for this block back to global memory
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}