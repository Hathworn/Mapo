#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void naive_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    extern __shared__ int shared_bins[];  // Declare shared memory for histogram bins

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;

    atomicAdd(&shared_bins[myBin], 1);   // Use shared memory to avoid global memory contention

    __syncthreads();  // Ensure all updates to shared memory are done

    if (threadIdx.x < BIN_COUNT) {
        atomicAdd(&d_bins[threadIdx.x], shared_bins[threadIdx.x]);  // Combine results back to global memory
    }
}