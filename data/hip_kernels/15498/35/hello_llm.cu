#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use threadIdx.x to identify individual threads within a block for detailed output
    printf("Hello world! I'm thread %d in block %d\n", threadIdx.x, blockIdx.x);
}