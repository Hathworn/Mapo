#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use register memory for thread index
    int tid = threadIdx.x; 

    // Utilize fast shared memory for frequently accessed data
    __shared__ char message[32]; 
    sprintf(message, "Hello world! I'm thread %d\n", tid);

    // Coalesced memory access by reducing printf calls
    printf("%s", message);
}