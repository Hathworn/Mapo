#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
{
    const float normalization_constant = 1.f / d_input_cdf[n - 1];
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for better performance on large data
    __shared__ unsigned int s_input_cdf[1024];

    if (global_index_1d < n)
    {
        s_input_cdf[threadIdx.x] = d_input_cdf[global_index_1d];
        __syncthreads();

        float output_value = s_input_cdf[threadIdx.x] * normalization_constant;
        d_output_cdf[global_index_1d] = output_value;
    }
}