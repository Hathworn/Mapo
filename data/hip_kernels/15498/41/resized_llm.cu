#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void resized(unsigned char *imgData, int width, float scale_factor, hipTextureObject_t texObj) {
    // Calculate global thread index
    const unsigned int tidX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int tidY = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned idx = tidY * width + tidX;

    // Ensure thread index is within bounds before accessing texture memory
    if (tidX < width && tidY < gridDim.y * blockDim.y) {
        // Read from texture memory and store in imgData
        imgData[idx] = tex2D<unsigned char>(texObj, tidX * scale_factor, tidY * scale_factor);
    }
}