#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
    extern __shared__ float shared_data[];
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // Load elements into shared memory
    shared_data[tid] = d_in[myId];
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block back to global memory
    if (tid == 0)
    {
        d_out[blockIdx.x] = shared_data[0];
    }
}