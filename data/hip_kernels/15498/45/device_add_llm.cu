#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_add(int *a, int *b, int *c) {
    // Pre-calculate and cache frequently accessed memory positions
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    // Utilize registers for temporary storage to reduce memory latency
    int valueA = a[index];
    int valueB = b[index];
    c[index] = valueA + valueB;
}