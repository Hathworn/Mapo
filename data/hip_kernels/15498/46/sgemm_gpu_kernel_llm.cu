#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_gpu_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < N && col < M) {  // Boundary check to prevent out-of-bounds memory access.
        float sum = 0;
        for (int i = 0; i < K; ++i) {
            sum += A[row * K + i] * B[i * M + col];  // Corrected index calculation for B.
        }
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}