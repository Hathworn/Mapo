#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth(float *v_new, const float *v) {
    // Calculate global thread index directly
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory to reduce global memory access
    extern __shared__ float shared[];
    int localIdx = threadIdx.x;
    
    // Load elements into shared memory
    shared[localIdx] = v[myIdx];
    
    if (localIdx > 0) {
        shared[localIdx - 1] = v[myIdx - 1];
    }
    
    if (localIdx < blockDim.x - 1) {
        shared[localIdx + 1] = v[myIdx + 1];
    }
    
    __syncthreads();
    
    // Calculate indices
    int numThreads = blockDim.x * gridDim.x;
    int myLeftIdx = (myIdx == 0) ? 0 : localIdx - 1;
    int myRightIdx = (myIdx == numThreads - 1) ? localIdx : localIdx + 1;
    
    // Compute new value using shared memory
    float myElt = shared[localIdx];
    float myLeftElt = shared[myLeftIdx];
    float myRightElt = shared[myRightIdx];
    v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
}