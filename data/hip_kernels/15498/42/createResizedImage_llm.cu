#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void createResizedImage(unsigned char *imageScaledData, int scaled_width, float scale_factor, hipTextureObject_t texObj)
{
    // Calculate a single index for threads
    const unsigned int tidX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int tidY = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int index = tidY * scaled_width + tidX;

    // Ensure the thread is within bounds
    if (tidX < scaled_width && tidY * scaled_width < scaled_width * gridDim.y * blockDim.y) {
        // Use tex2D to access texture memory
        imageScaledData[index] = tex2D<unsigned char>(texObj, tidX * scale_factor, tidY * scale_factor);
    }
}