#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Load thread index into a register for efficient access
    int tid = threadIdx.x;
    
    // Unroll loop if needed for repetitive operations (if applicable, here just a single printf)
    printf("Hello world! I'm thread %d\n", tid);
}