#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use shared memory for potential shared data, if needed in future
    __shared__ int sharedData; 

    // Minimize printf calls by using thread 0 of each block
    if (threadIdx.x == 0)
    {
        printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
    }
}