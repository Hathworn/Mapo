#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(float *d_out, float *d_in)
{
    // Use blockIdx.x for indexing to handle larger arrays
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Calculate the cube directly into the output array
    d_out[idx] = d_in[idx] * d_in[idx] * d_in[idx];
}