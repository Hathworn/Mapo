#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void idx_print() {
    // Calculate flattened thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate warp and lane indices
    int warp_idx = threadIdx.x / warpSize;
    int lane_idx = threadIdx.x & (warpSize - 1);

    // Print information only for lane 0 of each half-warp
    if((lane_idx & (warpSize/2 -1)) == 0) { 
        printf(" %5d\t%5d\t %2d\t%2d\n", idx, blockIdx.x, warp_idx, lane_idx);
    }
}