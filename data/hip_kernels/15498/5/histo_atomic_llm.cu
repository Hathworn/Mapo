#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_atomic(unsigned int *out_histo, const float *d_in, int num_bins, int size, float min_val, float range)
{
    // Calculate the global thread ID
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Exit if thread ID is out of bounds
    if (id >= size)
    {
        return;
    }

    // Calculate the histogram bin index
    int bin = ((d_in[id] - min_val) * num_bins) / range;
    bin = bin == num_bins ? num_bins - 1 : bin; // Ensure max value falls into the last bin

    // Atomically increment the histogram bin
    atomicAdd(&(out_histo[bin]), 1);
}