#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Use shared memory to reduce global memory contention
    extern __shared__ int shared_bins[];

    // Initialize shared memory
    int tid = threadIdx.x;
    if (tid < BIN_COUNT) {
        shared_bins[tid] = 0;
    }
    __syncthreads();

    // Compute histogram in shared memory
    int myId = tid + blockDim.x * blockIdx.x;
    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;
    atomicAdd(&(shared_bins[myBin]), 1);
    __syncthreads();

    // Write shared memory results back to global memory
    if (tid < BIN_COUNT) {
        atomicAdd(&(d_bins[tid]), shared_bins[tid]);
    }
}