#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;

    // Ensure myId does not exceed data bounds
    if (myId >= BIN_COUNT) return;

    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;

    // Use atomic function to ensure thread safety when updating shared data
    atomicAdd(&(d_bins[myBin]), 1);
}