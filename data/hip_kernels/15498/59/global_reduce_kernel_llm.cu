#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
    extern __shared__ float sdata[];  // Use shared memory for reduction
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // Load input into shared memory
    sdata[tid] = d_in[myId];
    __syncthreads(); // Make sure all threads have loaded their data

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();  // Ensure all additions at one stage are done
    }

    // Only thread 0 writes result for this block back to global memory
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}