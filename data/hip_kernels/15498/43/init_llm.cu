#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = index; i < n; i += stride) {
        x[i] = 1.0f; // Initialize x[i] with 1.0f
        y[i] = 2.0f; // Initialize y[i] with 2.0f
    }
}