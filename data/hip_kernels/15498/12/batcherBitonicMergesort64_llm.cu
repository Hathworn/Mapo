#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    // Optimize: Combine loops and add necessary synchronization
    for (int stage = 0; stage <= 5; stage++) {
        for (int substage = stage; substage >= 0; substage--) {
            int pairDistance = 1 << substage;
            int blockSize = 1 << (stage + 1);
            int comparePair = (tid / pairDistance) % 2 == 0;

            // Determine the indices to compare
            int compareIndex = tid ^ pairDistance;
            if (compareIndex < blockSize) {
                // Conditional swap if needed
                if ((comparePair && sdata[tid] > sdata[compareIndex]) || (!comparePair && sdata[tid] < sdata[compareIndex])) {
                    float temp = sdata[tid];
                    sdata[tid] = sdata[compareIndex];
                    sdata[compareIndex] = temp;
                }
            }
            __syncthreads(); // Ensure all swaps are finished before the next iteration
        }
    }

    d_out[tid] = sdata[tid];
}