#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(int n, float *x, float *y) {
    // Calculate lane id, warp id, and total warps
    int lane_id = threadIdx.x & 31;
    size_t warp_id = (threadIdx.x + blockIdx.x * blockDim.x) >> 5;
    size_t warps_per_grid = (blockDim.x * gridDim.x) >> 5;
    size_t warp_total = ((sizeof(float)*n) + STRIDE_64K-1) / STRIDE_64K;
    
    // Iterate over warps in grid, taking strides by warps_per_grid
    for(; warp_id < warp_total; warp_id += warps_per_grid) {
        #pragma unroll
        for(int rep = 0; rep < STRIDE_64K/sizeof(float)/32; rep++) {
            size_t ind = warp_id * STRIDE_64K/sizeof(float) + rep * 32 + lane_id;
            if (ind < n) {
                x[ind] = 1.0f;  // Set value in x
                y[ind] = 2.0f;  // Set value in y
            }
        }
    }
}
```
