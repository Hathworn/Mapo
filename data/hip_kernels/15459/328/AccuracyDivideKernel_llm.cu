#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Initialize thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Only thread 0 performs the division to avoid race conditions
    if (idx == 0) {
        *accuracy /= N;
    }
}