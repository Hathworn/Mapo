#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Use shared memory for better memory access patterns if needed
    extern __shared__ float shared_feats[];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single check to ensure we are within bounds
    if (row < num_rows && col < num_cols) {
        float f = feats[row * ldf + col];
        feats_sq[row * lds + col] = f * f; // Store squared value
    }
}