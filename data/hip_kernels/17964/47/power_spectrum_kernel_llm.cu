#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    // Calculate the offset using lane-specific indices
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure synchronization among threads using shared memory
    extern __shared__ float shared_mem[];

    int half_length = row_length / 2;
    
    // Loop through indices and optimize memory access
    for (int idx = thread_id; idx < half_length; idx += blockDim.x * gridDim.x) {
        // Ignore special case
        if (idx == 0) continue;

        // Prefetch data into shared memory
        float2 val = reinterpret_cast<float2 *>(A_in + blockIdx.x * ldi)[idx];
        shared_mem[threadIdx.x] = val.x * val.x + val.y * val.y;

        __syncthreads(); // Ensure all threads have computed their values

        if (use_power) {
            A_out[blockIdx.x * ldo + idx] = shared_mem[threadIdx.x];
        } else {
            A_out[blockIdx.x * ldo + idx] = sqrtf(shared_mem[threadIdx.x]);
        }
    }

    // Handle special case for index 0 separately
    if (threadIdx.x == 0) {
        float real = A_in[blockIdx.x * ldi];
        float im = A_in[blockIdx.x * ldi + row_length];

        if (use_power) {
            A_out[blockIdx.x * ldo] = real * real;
            A_out[blockIdx.x * ldo + half_length] = im * im;
        } else {
            A_out[blockIdx.x * ldo] = fabs(real);
            A_out[blockIdx.x * ldo + half_length] = fabs(im);
        }
    }
}