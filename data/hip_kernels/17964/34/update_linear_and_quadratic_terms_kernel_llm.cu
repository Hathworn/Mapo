#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel(int32_t n, float old_num_frames, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    float cur_weight = *cur_tot_weight;

    // Precompute common terms outside the loop
    float new_num_frames = old_num_frames + cur_weight;
    float old_prior_scale = (max_count != 0.0f) ? max(old_num_frames, (float)max_count) / max_count : 0.0f;
    float new_prior_scale = (max_count != 0.0f) ? max(new_num_frames, (float)max_count) / max_count : 0.0f;
    float prior_scale_change = 1.0f + new_prior_scale - old_prior_scale;

    int32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride = blockDim.x * gridDim.x;
    
    // Use a stride loop to replace original loop
    for (int32_t i = tid; i < n; i += stride) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        quadratic[diag_idx] += prior_scale_change;
    }

    // Perform atomic operation for safety in multi-thread environment
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        atomicAdd(&linear[0], prior_offset * prior_scale_change);
    }
}