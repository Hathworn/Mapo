#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function to utilize multiple threads
__global__ void add(int a, int b, int *c) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;  // Calculate unique thread ID
    if (tid == 0) {
        *c = a + b;  // Compute addition using the first thread only
    }
}