#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(int *a, int *b, int *c)
{
    // Optimize by using threadIdx.x to process elements in parallel
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}