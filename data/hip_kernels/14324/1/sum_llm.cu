#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(int *a, int *b, int *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global index
    if (i < n) // Check to prevent out-of-bounds access
    {
        c[i] = a[i] + b[i];
    }
}