#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 10

__global__ void sum(int *a, int *b, int *c)
{
    // Calculate global thread index directly
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) // Ensure index is within bounds
    {
        c[i] = a[i] + b[i];
    }
}