#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void block_sum(const int *input, int *per_block_results, const size_t n)
{
    extern __shared__ int sdata[];

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input into shared memory with bounds check
    int x = (i < n) ? input[i] : 0;
    sdata[threadIdx.x] = x;
    __syncthreads();

    // Unroll the loop for better performance
    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if(threadIdx.x < offset)
        {
            // Add a partial sum
            sdata[threadIdx.x] += sdata[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Write the result of the block's sum
    if(threadIdx.x == 0)
    {
        per_block_results[blockIdx.x] = sdata[0];
    }
}