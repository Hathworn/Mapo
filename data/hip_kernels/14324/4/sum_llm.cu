#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(int *a, int *b, int *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride outside the loop
    for (; i < N; i += stride) {         // Use for loop for clarity
        c[i] = a[i] + b[i];              // Perform sum operation
    }
}