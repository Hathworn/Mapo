#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to add the elements of two arrays

__global__ void add(int n, float *x, float *y)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Optimize memory access by iterating over the global stride
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}