#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float3 color(unsigned int depth, unsigned int maxDepth)
{
    if(depth == maxDepth)
        return make_float3(0.0f, 0.0f, 0.0f);
    else
        return make_float3(1.0f, 1.0f, 1.0f);
}

__device__ unsigned int mandelDepth(float cr, float ci, int maxDepth)
{
    float zr = 0.0f;
    float zi = 0.0f;
    float zrSqr = 0.0f;
    float ziSqr = 0.0f;

    unsigned int i;

    for (i = 0; i < maxDepth; i++)
    {
        zi = zr * zi;
        zi += zi + ci;
        zr = zrSqr - ziSqr + cr;
        zrSqr = zr * zr;
        ziSqr = zi * zi;

        if (zrSqr + ziSqr > 4.0f) break;
    }

    return i;
}

__global__ void mandel(float* buffer, float xMin, float xMax, float yMin, float yMax, unsigned int maxDepth)
{
    // Calculate indices and steps
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int nx = blockDim.x * gridDim.x;
    int ny = gridDim.x;
    float dx = (xMax - xMin) / nx;
    float dy = (yMax - yMin) / ny;

    // Calculate position in complex plane
    float x = xMin + (idx % nx + 0.5f) * dx;
    float y = yMin + (blockIdx.x + 0.5f) * dy;

    // Compute Mandelbrot set depth
    unsigned int depth = mandelDepth(x, y, maxDepth);

    // Get color for the depth
    float3 depthColor = color(depth, maxDepth);

    // Write color to buffer
    buffer[3*idx + 0] = depthColor.x;
    buffer[3*idx + 1] = depthColor.y;
    buffer[3*idx + 2] = depthColor.z;
}