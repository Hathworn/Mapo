#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate the global index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll the loop by 4 to increase performance
    unsigned int stride = blockDim.x * gridDim.x;
    for (unsigned int i = idx; i < numEls; i += stride * 4) {
        if (i < numEls) dest[i] = a[i] + b[i];
        if (i + stride < numEls) dest[i + stride] = a[i + stride] + b[i + stride];
        if (i + 2 * stride < numEls) dest[i + 2 * stride] = a[i + 2 * stride] + b[i + 2 * stride];
        if (i + 3 * stride < numEls) dest[i + 3 * stride] = a[i + 3 * stride] + b[i + 3 * stride];
    }
}