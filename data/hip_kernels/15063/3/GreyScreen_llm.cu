#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GreyScreen(float* d_pixelsR, float* d_pixelsG, float* d_pixelsB, float* d_reducePixels, int numPixels) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Check bounds to ensure valid access
    if (id < numPixels) {
        // Compute the average directly for greyscale
        d_reducePixels[id] = (d_pixelsR[id] + d_pixelsG[id] + d_pixelsB[id]) / 3.0f;
    }
}