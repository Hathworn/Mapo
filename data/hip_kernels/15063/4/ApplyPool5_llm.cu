#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ApplyPool5(float* input, float* output){
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Loop unrolling to reduce loop overhead
    for (int i = 0; i < 148; i += 2){
        // Using local variables to reduce redundant memory access
        float input1 = input[i * 2 + id * 2 * 296];
        float input2 = input[i * 2 + id * 2 * 296 + 1];
        float input3 = input[i * 2 + id * 2 * 296 + 296];
        float input4 = input[i * 2 + id * 2 * 296 + 296 + 1];
        
        float input5 = input[(i + 1) * 2 + id * 2 * 296];
        float input6 = input[(i + 1) * 2 + id * 2 * 296 + 1];
        float input7 = input[(i + 1) * 2 + id * 2 * 296 + 296];
        float input8 = input[(i + 1) * 2 + id * 2 * 296 + 296 + 1];

        // Compute total using max operation
        float total1 = max(max(input1, input2), max(input3, input4));
        float total2 = max(max(input5, input6), max(input7, input8));

        // Add activation function
        total1 = 1 / (1 + exp(-(total1 * 2 - 1)));
        total2 = 1 / (1 + exp(-(total2 * 2 - 1)));

        // Store the result
        output[i + id * 148] = total1;
        output[i + 1 + id * 148] = total2;
    }
}