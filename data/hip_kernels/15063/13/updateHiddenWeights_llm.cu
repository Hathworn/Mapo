#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateHiddenWeights(float* d_weights, float error, float lr, int keyPress, float* d_outputweights, int screenSize, int numHiddenNeurons, float* d_bias, float* firstFire){
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    if (id >= numHiddenNeurons) return;  // Exit if thread index is out of bounds

    float totalChange = 0.0f;
    for (int i = 0; i < screenSize; ++i) {
        // Cache repeated calculations
        float outputWeight = d_outputweights[id * numHiddenNeurons + keyPress];
        float fireEffect = firstFire[id] * 2 - 1;
        float change = error * lr * outputWeight * fireEffect;
        totalChange += change;

        // Combined min and max operations
        float newWeight = d_weights[id * screenSize + i] + change;
        d_weights[id * screenSize + i] = fminf(1.0f, fmaxf(0.0f, newWeight));
    }
    float biasChange = totalChange * -0.5f;
    d_bias[id] += biasChange;
}