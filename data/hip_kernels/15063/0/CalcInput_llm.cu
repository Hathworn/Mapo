#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CalcInput(float* screen, float* weight, float* d_Votes, int stride) {

    // Calculate the global index for the current thread
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    // Initialize the vote for this index to zero
    float vote = 0.0f;

    // Combine operations to reduce redundant memory access
    vote += screen[id] * weight[id];
    vote += screen[id + 1] * weight[id + 1];
    vote += screen[stride] * weight[stride];
    vote += screen[stride + 1] * weight[stride + 1];

    // Normalize the vote by averaging
    d_Votes[id] = vote * 0.25f;  // Use multiplication instead of division for efficiency
}