#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FirstHidden(float* input, float* weight, float* bias, int d_numVotes, float* d_votes) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    float total = bias[id]; // Initialize total with bias value

    for (int i = 0; i < d_numVotes; ++i) {
        total += input[i] * weight[id * d_numVotes + i]; // Accumulate weighted input
    }

    d_votes[id] = 1.0f / (1.0f + expf(-total)); // Apply sigmoid function to total
}