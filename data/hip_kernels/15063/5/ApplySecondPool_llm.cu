#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ApplySecondPool(float* input, float* output) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Cache reused index calculations into variables
    int baseIdx = id * 2 * 146;
    int outputBaseIdx = id * 73;

    for (int i = 0; i < 73; ++i) {
        // Use local variables for indices to improve readability
        int inputBaseIdx = i * 2 + baseIdx;
        
        // Calculate maximum using a single step with a temporary variable
        float maxVal = max(input[inputBaseIdx], input[inputBaseIdx + 1]);
        maxVal = max(maxVal, input[inputBaseIdx + 146]);
        maxVal = max(maxVal, input[inputBaseIdx + 147]);

        // Optimized output calculation
        output[outputBaseIdx + i] = 1 / (1 + exp(-(maxVal * 2 - 1)));
    }
}