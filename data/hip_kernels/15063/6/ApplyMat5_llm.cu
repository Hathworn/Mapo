#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ApplyMat5(float* input, float* output, float* matrix) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    // Loop unrolling and memory access optimization
    for (int i = 0; i < 296; ++i) {
        float total = 0.0f;

        #pragma unroll
        for (int row = 0; row < 5; ++row) {
            for (int col = 0; col < 5; ++col) {
                // Reduce index computation in loop
                total += input[id * 300 + i + 300 * row + col] * matrix[row * 5 + col];
            }
        }

        // Apply ReLU activation
        total = fmax(0.0f, total);

        // Store result in the output
        output[i + id * 296] = total;
    }
}