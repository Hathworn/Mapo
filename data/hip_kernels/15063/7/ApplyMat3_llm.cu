#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ApplyMat3(float* input, float* output, float* matrix){
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    __shared__ float smatrix[9];  // Allocate shared memory to store matrix
    if(threadIdx.x < 9) {
        smatrix[threadIdx.x] = matrix[threadIdx.x];  // Load matrix into shared memory
    }
    __syncthreads();  // Ensure all threads have loaded the matrix

    for (int i = 0; i < 146; ++i){
        float total = 0.0f;

        int baseIdx = id * 148 + i;
        total += input[baseIdx] * smatrix[0];
        total += input[baseIdx + 1] * smatrix[1];
        total += input[baseIdx + 2] * smatrix[2];

        total += input[baseIdx + 148] * smatrix[3];
        total += input[baseIdx + 148 + 1] * smatrix[4];
        total += input[baseIdx + 148 + 2] * smatrix[5];

        total += input[baseIdx + 296] * smatrix[6];
        total += input[baseIdx + 296 + 1] * smatrix[7];
        total += input[baseIdx + 296 + 2] * smatrix[8];

        total = fmaxf(0.0f, total);  // Use fmaxf for single precision

        output[i + id * 146] = total;
    }
}