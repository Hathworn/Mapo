#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void OutputLayer(float* hiddenVotes, float* weight, int d_numHiddenNodes, float* d_votes) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    // Use register for the accumulator
    register float total = 0.0f;
    int weightIndex = id * d_numHiddenNodes;

    // Use loop unrolling for better performance
    for (int i = 0; i < d_numHiddenNodes - 3; i += 4) {
        total += hiddenVotes[i] * weight[weightIndex + i];
        total += hiddenVotes[i + 1] * weight[weightIndex + i + 1];
        total += hiddenVotes[i + 2] * weight[weightIndex + i + 2];
        total += hiddenVotes[i + 3] * weight[weightIndex + i + 3];
    }
    // Handle remainder of the loop
    for (int i = (d_numHiddenNodes / 4) * 4; i < d_numHiddenNodes; ++i) {
        total += hiddenVotes[i] * weight[weightIndex + i];
    }

    d_votes[id] = total;
}