#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateOutputWeights(float* d_weights, float error, float lr, int keypress, int numHiddenNeurons, float* outputTotals, int numInput) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    // Calculate index only if id is valid
    if (id < numHiddenNeurons) {
        int index = numHiddenNeurons * keypress + id;

        // Calculate certainty in parallel using shared memory.
        __shared__ float shared_outputTotals[1024]; // Assume max numInput is 1024
        if (id < numInput) {
            shared_outputTotals[id] = outputTotals[id];
        }
        __syncthreads();

        float certainty = 0.0f;
        for (int i = 0; i < numInput; ++i) {
            certainty += shared_outputTotals[i];
        }
        certainty = shared_outputTotals[keypress] / certainty; // Reduce memory access

        // Calculate change and update weight
        float change = error * lr * d_weights[index] * certainty;
        d_weights[index] += change;

        // Clamp weights using fminf and fmaxf for efficiency
        d_weights[index] = fminf(1.0f, d_weights[index]);
        d_weights[index] = fmaxf(0.0f, d_weights[index]);
    }
}