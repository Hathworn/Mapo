#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateInput(float* screen, float* weight, float* d_Votes) {
    // Calculate thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit if index is out of bounds
    if (idx >= N) return;

    // Load data with coalesced memory access
    float vote = d_Votes[idx];
    float w = weight[idx];

    // Perform the update operation
    screen[idx] = w * vote;
}