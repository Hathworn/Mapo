#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CombineScreen(float* d_postEdge1, float* d_postEdge2, float* d_postGradient1, float* d_postGradient2, float* d_postGradient3, float* d_postSobel3LR, float* d_postSobel3UD, float* d_postSmooth31, float* d_output){
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int offset = id * 73;

    // Use register storage to reduce redundant calculations
    float* edge1 = d_postEdge1 + offset;
    float* edge2 = d_postEdge2 + offset;
    float* gradient1 = d_postGradient1 + offset;
    float* gradient2 = d_postGradient2 + offset;
    float* gradient3 = d_postGradient3 + offset;
    float* sobel3LR = d_postSobel3LR + offset;
    float* sobel3UD = d_postSobel3UD + offset;
    float* smooth31 = d_postSmooth31 + offset;

    for (int i = 0; i < 73; ++i){
        int outIdx = i + offset;
        d_output[outIdx + 0 * 73 * 73] = edge1[i];
        d_output[outIdx + 1 * 73 * 73] = edge2[i];
        d_output[outIdx + 2 * 73 * 73] = gradient1[i];
        d_output[outIdx + 3 * 73 * 73] = gradient2[i];
        d_output[outIdx + 4 * 73 * 73] = gradient3[i];
        d_output[outIdx + 5 * 73 * 73] = sobel3LR[i];
        d_output[outIdx + 6 * 73 * 73] = sobel3UD[i];
        d_output[outIdx + 7 * 73 * 73] = smooth31[i];
    }
}