#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function for adding two numbers
__global__ void addition(int a, int b, int *c)
{
    // Calculate unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform addition only for the first thread to minimize divergence
    if (idx == 0)
    {
        *c = a + b;
    }
}