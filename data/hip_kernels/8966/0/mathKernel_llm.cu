#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Lets you use the Cuda FFT library

hipError_t mathWithCuda(float *output, float *input1, float *input2, unsigned int size, int oper);

// Using __global__ to declare function as device code (GPU)
// Do the math inside here:

// Helper function for using CUDA to add vectors in parallel.
__global__ void mathKernel(float *output, float *input1, float *input2, int n, int oper)
{
    // Allocate elements to threads
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // Avoid access beyond the end of the array
    if (i < n)
    {
        // Simplified operations using conditional statements
        float val1 = input1[i];
        float val2 = input2[i];
        
        if (oper == 1) output[i] = val1 + val2; // Addition
        else if (oper == 2) output[i] = val1 - val2; // Subtraction
        else if (oper == 3) output[i] = val1 * val2; // Multiplication
        else if (oper == 4) output[i] = val1 / val2; // Division
    }
}