#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    // Set pointers to the appropriate position in the arrays
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;

    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // Compute partial dot products in parallel
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Perform parallel reduction to sum up the dot products
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }

    // Use the reduction result to compute the gradient
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}