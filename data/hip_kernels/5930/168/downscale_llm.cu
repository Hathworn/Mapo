#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii /= d3;
    z = ii % d2;
    ii /= d2;
    y = ii % d1;
    ii /= d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii /= d3;
    z = ii % d2;
    ii /= d2;
    y = ii % d1;
    ii /= d1;
    x = ii;
    w /= scale_factor;
    z /= scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x;
    ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
    if (ii >= no_elements) return;
    
    // Calculate stride once and use in loop
    int stride_d2 = d2 * scale_factor;
    int stride_d3 = d3 * scale_factor;

    for (int i = 0; i < scale_factor; ++i) {
        for (int j = 0; j < scale_factor; ++j) {
            int ipidx = translate_idx_inv(ii, d1, stride_d2, stride_d3, scale_factor, i, j);
            gradInput_data[ii] += gradOutput_data[ipidx];
        }
    }
}