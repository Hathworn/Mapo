#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Avoid redundant computation by using temporary variable
    float acc = *accuracy;
    acc /= N; 
    *accuracy = acc;
}