#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256  // Define threads per block for optimal performance

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use threadIdx.x to parallelize computation and improve performance
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        // Use atomic operation to ensure safe division update for accuracy
        atomicExch(accuracy, *accuracy / N);
    }
}