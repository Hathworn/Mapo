#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_WindowGate_updateOutput_kernel(float *output, float *centroids, float *normalizedCentroids, float *outputIndice, const float *input, const float *noise, int inputSize, int outputSize, int outputWindowSize, float a, float b, int train)
{
    __shared__ float buffer[WINDOWGATE_THREADS];
    unsigned int tx = threadIdx.x;
    unsigned int k = blockIdx.x;
    const float *input_k = input + inputSize * k;
    float *output_k = output + outputWindowSize * k;

    // Initialize shared memory buffer
    buffer[tx] = 0.0f;

    // Parallel reduction to compute centroid
    for (unsigned int i = tx; i < inputSize; i += blockDim.x)
    {
        buffer[tx] += input_k[i] * static_cast<float>(i + 1);
    }

    // Synchronize threads and perform reduction
    __syncthreads();
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tx < stride)
        {
            buffer[tx] += buffer[tx + stride];
        }
        __syncthreads();
    }

    // Calculate centroid and process it
    float centroid;
    if (tx == 0)
    {
        centroid = buffer[0] / static_cast<float>(inputSize);
        normalizedCentroids[k] = centroid;
        if (train)
        {
            centroid += noise[k];
            centroid = fminf(fmaxf(centroid, 0.0f), 1.0f);
        }
        centroid *= static_cast<float>(outputSize);

        float outputIdx = centroid - 0.5f * static_cast<float>(outputWindowSize);
        outputIdx = fminf(fmaxf(1.0f, outputIdx), outputSize - outputWindowSize + 1);
        outputIdx = ceilf(outputIdx);
        centroid -= (outputIdx - 1);

        outputIndice[k] = static_cast<int>(outputIdx);
        centroids[k] = centroid;
        buffer[0] = centroid;
    }
    
    __syncthreads();

    // Gaussian blur calculation
    centroid = buffer[0];
    for (int i = tx; i < outputWindowSize; i += blockDim.x)
    {
        float x = static_cast<float>(i + 1) - centroid;
        output_k[i] = a * expf(x * x * b);
    }
}