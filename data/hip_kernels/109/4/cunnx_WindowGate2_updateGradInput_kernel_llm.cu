#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunnx_WindowGate2_updateGradInput_kernel( 
    float *gradInput, 
    float *error, 
    float *targetCentroids, 
    const float *centroids, 
    const float *input, 
    const float *inputIndice, 
    const float *outputIndice, 
    const float *output, 
    const float *gradOutput, 
    int inputSize, 
    int outputSize, 
    int inputWindowSize, 
    int outputWindowSize, 
    int windowStride, 
    float c, 
    float d, 
    float e, 
    float lr) 
{
    unsigned int tx = threadIdx.x;
    unsigned int k = blockIdx.x;
    const float *gradOutput_k = gradOutput + outputWindowSize * k;
    float *gradInput_k = gradInput + inputSize * k;

    float *gradInputWindow = gradInput_k + (int)(inputIndice[k] - 1);

    // Optimize memory access by using shared memory
    __shared__ float sharedGradOutput[256]; // Adjust size as needed

    for (int i = tx; i < inputWindowSize; i += blockDim.x) 
    {
        float sum = 0;
        const float *gradOutputChannel = gradOutput_k + i * windowStride;
        
        // Copy gradOutputChannel to shared memory
        for (int j = 0; j < windowStride; j += blockDim.x) 
        {
            sharedGradOutput[j + tx] = gradOutputChannel[j + tx];
        }
        
        // Synchronize to ensure all threads have completed memory transfer
        __syncthreads();

        // Use shared memory for computation
        for (int j = 0; j < windowStride; j++) 
        {
            sum += sharedGradOutput[j];
        }

        gradInputWindow[i] += sum;

        // Synchronize to ensure memory consistency
        __syncthreads();
    }
}