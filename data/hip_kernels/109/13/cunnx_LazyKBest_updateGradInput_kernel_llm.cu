#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunnx_LazyKBest_updateGradInput_kernel(float *gradInput, const float *indice, const float *gradOutput, int inputSize, int outputSize)
{
    // Use shared memory for coalesced access
    __shared__ float s_gradOutput[1024];
    __shared__ float s_indice[1024];

    int tx = threadIdx.x;
    int k = blockIdx.x;

    float *gradInput_k = gradInput + k * inputSize;
    const float *gradOutput_k = gradOutput + k * outputSize;
    const float *indice_k = indice + k * outputSize;

    // Load gradOutput and indice into shared memory
    for (int i = tx; i < outputSize; i += blockDim.x) {
        s_gradOutput[i] = gradOutput_k[i];
        s_indice[i] = indice_k[i];
    }
    __syncthreads();

    // Use shared memory for writing gradInput
    for (int i = tx; i < outputSize; i += blockDim.x) {
        gradInput_k[(int)(s_indice[i] - 1)] = s_gradOutput[i];
    }
}