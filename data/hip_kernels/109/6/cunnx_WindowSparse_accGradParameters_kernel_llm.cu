#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunnx_WindowSparse_accGradParameters_kernel(float *gradWeight, float* gradBias, float *gradOutput, float *input, float *inputIndice, float *outputIndice, int inputWindowSize, int outputWindowSize, int inputSize, int outputSize, float scale) {
    __shared__ float buffer[WINDOWSPARSE_THREADS];
    int tx = threadIdx.x;
    int i_step = blockDim.x;
    int k = blockIdx.x;

    int inputIdx = (int)inputIndice[k] - 1;
    int outputIdx = (int)outputIndice[k] - 1;

    float *input_k = input + k * inputWindowSize;
    float *gradOutput_k = gradOutput + k * outputWindowSize;
    float *gradWeight_k = gradWeight + outputIdx * inputSize + inputIdx;
    float *gradBias_k = gradBias + outputIdx;

    // Coalesced memory access pattern for input
    for (int i = tx; i < inputWindowSize; i += i_step) {
        buffer[tx] = input_k[i] * scale;  // Efficient memory access and storing scaled input

        // Loop unrolling for outputWindowSize
        for (int j = 0; j < outputWindowSize; j++) {
            // Use shared memory buffer for intermediate computation
            atomicAdd(&(gradWeight_k[j * inputSize + i]), gradOutput_k[j] * buffer[tx]);
        }
    }

    // Bias update with efficient atomic operations
    for (int j = tx; j < outputWindowSize; j += i_step) {
        atomicAdd(&(gradBias_k[j]), gradOutput_k[j] * scale);
    }
}