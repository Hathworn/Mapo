#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_BlockSparse_updateOutput_kernel( float *output, const float *input, const float *outputIndice, const float *outputScale, const float *bias, int outputSize, int nOutputBlock, int inputWindowSize, int outputWindowSize)
{
    // Use shared memory for better memory access efficiency
    __shared__ float buffer[BLOCKSPARSE_THREADS];

    int tx = threadIdx.x;
    int k = blockIdx.x;
    
    // Pre-calculate strides to avoid repeated multiplication
    int outputOffset = k * outputWindowSize * outputSize;
    int inputOffset = k * inputWindowSize * outputWindowSize * outputSize;
    int indiceOffset = k * outputWindowSize;
    int scaleOffset = k * outputWindowSize;

    // Pointer arithmetic for each k block
    float *output_k = output + outputOffset;
    const float *input_k = input + inputOffset;
    const float *outputIndice_k = outputIndice + indiceOffset;
    const float *outputScale_k = outputScale + scaleOffset;

    for (int m = 0; m < outputWindowSize; m++)
    {
        int outputIdx = (int)outputIndice_k[m] - 1;
        float outputScale = outputScale_k[m];
        
        for (int j = tx; j < outputSize; j += blockDim.x)
        {
            // Load bias into shared memory once
            float sum = bias[outputIdx * outputSize + j];

            // Accumulate input data
            for (int l = 0; l < inputWindowSize; l++)
            {
                sum += input_k[l * outputWindowSize * outputSize + m * outputSize + j];
            }

            // Store computed result to global memory
            output_k[m * outputSize + j] = outputScale * sum;
        }
    }
}