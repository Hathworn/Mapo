#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_WindowGate2_updateOutput_kernel( float *output, float *centroids, float *normalizedCentroids, float *inputIndice, float *outputIndice, const float *input, const float *noise, int inputSize, int outputSize, int inputWindowSize, int outputWindowSize, int windowStride, int train)
{
    __shared__ float buffer[WINDOWGATE2_THREADS+1];
    unsigned int tx = threadIdx.x;
    unsigned int k = blockIdx.x;
    const float *input_k = input + inputSize*k;
    float *output_k = output + outputWindowSize*k;
    
    // Initialize shared memory
    buffer[tx] = 0;

    // Sum up the weighted inputs
    for (unsigned int i = tx; i < inputSize; i += blockDim.x)
        buffer[tx] += input_k[i] * (float)(i + 1);

    // Parallel reduction
    __syncthreads();
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
        if (tx < stride)
            buffer[tx] += buffer[tx + stride];
        __syncthreads();
    }

    if (tx == 0) {
        float centroid = buffer[0] / (float)(inputSize);

        normalizedCentroids[k] = centroid;
        if (train) {
            centroid += noise[k];
            centroid = fminf(fmaxf(0, centroid), 1);
        }

        // Align centroid to output
        centroid *= (float)(outputSize);

        float inputIdx = centroid / (float)(inputSize) - 0.5f * (float)inputWindowSize;
        float outputIdx = centroid - 0.5f * (float)outputWindowSize;

        // Clip indices
        inputIdx = fminf(fmaxf(ceilf(fminf(inputIdx, (float)(inputSize - inputWindowSize + 1))), 1), (float)(inputSize));
        outputIdx = fminf(fmaxf(ceilf(fminf(outputIdx, (float)(outputSize - outputWindowSize + 1))), 1), (float)(outputSize));

        // Align centroid to output window
        centroid -= (outputIdx - 1);

        inputIndice[k] = (int)inputIdx;
        outputIndice[k] = (int)outputIdx;
        centroids[k] = centroid;

        buffer[WINDOWGATE2_THREADS] = inputIdx;
    }

    __syncthreads();

    // Load input window
    float inputIdx = buffer[WINDOWGATE2_THREADS];
    const float *inputWindow = input_k + (int)inputIdx;

    // Fill output window
    for (int i = tx; i < outputWindowSize; i += blockDim.x) {
        output_k[i] = inputWindow[(int)floorf(((float)i) / windowStride)];
    }
}