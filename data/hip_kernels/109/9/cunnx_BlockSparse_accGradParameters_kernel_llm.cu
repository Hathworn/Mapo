#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_BlockSparse_accGradParameters_kernel( float *gradWeight, float* gradBias, float *gradOutput, float *input, float *inputIndice, float *outputIndice, int inputSize, int outputSize, int nInputBlock, int nOutputBlock, int inputWindowSize, int outputWindowSize, float scale)
{
    __shared__ float buffer[BLOCKSPARSE_THREADS];
    __shared__ float gradOutputBuffer[BLOCKSPARSE_MAXOUTPUTBLOCKSIZE];
    int tx = threadIdx.x;
    int i_step = blockDim.x;
    int k = blockIdx.x;

    float *input_k = input + k*inputWindowSize*inputSize;
    float *gradOutput_k = gradOutput + k*outputWindowSize*outputSize;
    float *inputIndice_k = inputIndice + k*inputWindowSize;
    float *outputIndice_k = outputIndice + k*outputWindowSize;

    // Loop through blocks
    for (int m=0; m<outputWindowSize; m++)
    {
        int outputIdx = (int)outputIndice_k[m] - 1;

        float *blockGradOutput = gradOutput_k + m*outputSize;
        float *blockGradBias = gradBias + outputIdx*outputSize;

        for (int j=tx; j<outputSize; j+=i_step)
            gradOutputBuffer[j] = blockGradOutput[j]*scale;

        __syncthreads(); // Synchronize to ensure gradOutputBuffer is ready

        for (int l=0; l<inputWindowSize; l++)
        {
            int inputIdx = (int)inputIndice_k[l] - 1;

            float *blockInput = input_k + l*inputSize;
            float *blockGradWeight = gradWeight + outputIdx*nInputBlock*outputSize*inputSize + inputIdx*outputSize*inputSize;

            // Process input blocks in parallel
            buffer[tx] = blockInput[tx];  // Load input to buffer for current thread

            // Multiply accumulate weights
            for (int i=tx; i<inputSize; i+=i_step)
            {
                if (i < inputSize)
                {
                    for (int j=0; j<outputSize; j++)
                    {
                        atomicAdd(&(blockGradWeight[j*inputSize + i]), gradOutputBuffer[j]*buffer[tx]);
                    }
                }
            }
        }

        __syncthreads(); // Synchronize for consistent bias updates

        // Multiply accumulate biases
        for (int j=tx; j<outputSize; j+=i_step)
            atomicAdd(&(blockGradBias[j]), gradOutputBuffer[j]);
    }
}