#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multKernel(int *c, const int *a, const int *b)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Perform multiplication only if within array bounds
    if (i < N) {
        c[i] = a[i] * b[i];
    }
}