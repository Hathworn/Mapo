#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multKernel(int *c, const int *a, const int *b)
{
    // Use blockIdx and blockDim for better scalability
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid magic numbers, use a constant instead
    const int multiplier = 100;
    c[i] = a[i] * b[i] * multiplier;
}