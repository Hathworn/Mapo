#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define DEBUG 0

__global__ void jacobiOnGpuNaive2D(double *MatA, double *d_MatC, int dim_x, int dim_y, int iter_max) {
    unsigned long long int i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned long long int j = blockDim.y * blockIdx.y + threadIdx.y * 4;
    unsigned long long int index = i * dim_y + j;

    // Cache boundary conditions
    bool within_bounds = (i > 0 && j > 0 && i < (dim_x - 1) && j < (dim_y - 1));

    for (int k = 0; k < iter_max; k++) {
        if (within_bounds) {
            d_MatC[index] = 0.25 * (MatA[(i-1) * dim_y + j] + MatA[(i+1) * dim_y + j] + 
                                    MatA[i * dim_y + (j-1)] + MatA[i * dim_y + (j+1)]);
            // Reduce index calculation by using local variable with unrolling
            for (int offset = 1; offset <= 3; ++offset) {
                unsigned long long int idx = index - offset;
                d_MatC[idx] = 0.25 * (MatA[idx + 1] + MatA[idx - 1] + 
                                      MatA[idx + dim_y] + MatA[idx - dim_y]);
            }

            if (DEBUG) {
                printf("index %llu %llu %llu %llu\n", index, index - 3, index - 2, index - 1);
            }
        }
        __syncthreads();
        if (index < dim_x * dim_y) MatA[index] = d_MatC[index];
    }
}