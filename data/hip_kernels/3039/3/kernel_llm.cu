#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float fx(float a, float b) {
    return a + b;
}

__global__ void kernel(void) {
    // Avoid unnecessary device function call by inlining
    float result = 1.0 + 2.0;
    printf("res = %f\n", result);
}