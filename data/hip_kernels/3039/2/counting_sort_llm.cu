#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void counting_sort(int* array, int *temp, int size) {
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    if (i < size) {
        int count = 0;
        int ai = array[i]; // Load array[i] once
        for(int j = 0; j < size; j++) {
            int aj = array[j]; // Load array[j] once
            if(aj < ai) {
                count++;
            } else if(ai == aj && j < i) {
                count++;
            }
        }
        temp[count] = ai;
    }
}