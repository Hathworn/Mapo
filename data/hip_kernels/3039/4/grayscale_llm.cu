#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void grayscale(unsigned char *src, unsigned char *dest, int width, int height, int nChannels) {
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure threads processing valid pixels
    if (y >= height || x >= width) return;
    
    int pos = (y * width + x) * nChannels;
    
    // Use registers to store pixel values
    float r = src[pos + 2];
    float g = src[pos + 1];
    float b = src[pos + 0];
    
    // Pre-compute for efficiency and apply fminf to ensure max value is 255
    float tr = fminf(0.393f * r + 0.769f * g + 0.189f * b, 255.0f);
    float tg = fminf(0.349f * r + 0.686f * g + 0.168f * b, 255.0f);
    float tb = fminf(0.272f * r + 0.534f * g + 0.131f * b, 255.0f);
    
    dest[pos + 2] = static_cast<unsigned char>(tr);
    dest[pos + 1] = static_cast<unsigned char>(tg);
    dest[pos + 0] = static_cast<unsigned char>(tb);
}