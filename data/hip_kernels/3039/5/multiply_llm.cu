#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiply(int* a, int* b, int* c, int x, int y) {
    // Calculate global row and column indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < x && col < x) { // Check bounds
        int temp = 0; // Initialize result accumulator
        
        // Use register for intermediate results to reduce global memory accesses
        for(int i = 0; i < y; i++) {
            temp += a[row * y + i] * b[i * x + col];
        }
        
        c[row * x + col] = temp; // Store result
    }
}