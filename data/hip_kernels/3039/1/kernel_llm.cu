#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Define maximum number of simultaneous threads to reduce serialization.
#define MAX_THREADS_PER_BLOCK 256

__global__ void kernel(void) {
    // Use minimized index calculations and reduce print frequency to lessen I/O overhead.
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Conditional statement to limit excessive printing.
    if (globalIndex % MAX_THREADS_PER_BLOCK == 0) {
        printf("GPU bockIdx %i threadIdx %i: Hello World!\n", blockIdx.x, threadIdx.x);
    }
}