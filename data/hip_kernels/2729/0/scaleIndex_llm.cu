#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#define TILE 16

__global__ void scaleIndex(double *matrix, int n, int index){
    // Calculate the thread's position within the row
    int thread_idx = threadIdx.x + blockDim.x * blockIdx.x;

    // Calculate the starting and ending positions for this row
    int start = index * n + index;
    int end = index * n + n;

    // Loop with stride to handle large matrices
    for (int i = start + 1 + thread_idx; i < end; i += blockDim.x * gridDim.x) {
        matrix[i] = (matrix[i] / matrix[start]);
    }
}