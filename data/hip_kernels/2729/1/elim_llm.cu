#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#define TILE 16

__global__ void elim(double *A, int n, int index, int bsize) {
    extern __shared__ double pivot[];

    int idThread = threadIdx.x;
    int idBlock = blockIdx.x;
    int blockSize = bsize;
    
    // Load pivot row into shared memory by first thread in block
    if (idThread == 0) {
        for (int i = index; i < n; i++) {
            pivot[i] = A[index * n + i];
        }
    }

    __syncthreads();

    // Calculate current row and start indices
    int currentRow = ((blockSize * idBlock) + idThread) * n;
    int start = currentRow + index;

    // Perform elimination if the current row is below the pivot row
    if (currentRow > index * n) {
        double startVal = A[start];
        for (int i = start + 1; i < currentRow + n; ++i) {
            A[i] -= startVal * pivot[i - currentRow];
        }
    }
}