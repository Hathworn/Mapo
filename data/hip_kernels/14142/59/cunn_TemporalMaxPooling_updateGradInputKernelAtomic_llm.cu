#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate data pointer offsets once outside the loop
    int threadIndex = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    if (threadIndex >= output_w) return; // Terminate if index is out-of-bounds

    // Pre-calculate pointers
    float *gradInput_data = gradInput + blockIdx.x * input_w * input_n + threadIndex * input_n * dW;
    float *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + threadIndex * input_n;
    float *indices_data = indices + blockIdx.x * output_w * input_n + threadIndex * input_n;

    // Parallelize feature updates using thread parallelism
    for (int feat = threadIdx.y; feat < input_n; feat += blockDim.y) {
        atomicAdd(&gradInput_data[(int)indices_data[feat] * input_n + feat], gradOutput_data[feat]);
    }
}