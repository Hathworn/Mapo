#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor)
{
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void upscale(float *input, float *output, long no_elements, int scale_factor, int d1, int d2, int d3)
{
    // Calculate the linear index using 3D blocks and threads
    long ii = blockIdx.x * blockDim.x + threadIdx.x;
    long blockIdyOffset = blockIdx.y * (blockDim.x * gridDim.x);
    long blockIdzOffset = blockIdx.z * (blockDim.x * gridDim.x * gridDim.y);
    ii += blockIdx.y * blockDim.y + threadIdx.y + blockIdyOffset;
    ii += blockIdx.z * blockDim.z + threadIdx.z + blockIdzOffset;

    // Boundary check
    if (ii >= no_elements) return;

    // Calculate source index and copy to output
    int ipidx = translate_idx(ii, d1, d2, d3, scale_factor);
    output[ii] = input[ipidx];
}