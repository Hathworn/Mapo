#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate global index for threads
    int globalIndex = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    
    // Check if within output width to reduce divergence
    if (globalIndex < output_w) {
        // Calculate offsets for gradInput, gradOutput, and indices
        size_t batchOffset = blockIdx.x * input_n;
        float *gradInput_data = gradInput + batchOffset * input_w + globalIndex * input_n * dW;
        float *gradOutput_data = gradOutput + batchOffset * output_w + globalIndex * input_n;
        float *indices_data = indices + batchOffset * output_w + globalIndex * input_n;

        // Process all features
        for (int feat = 0; feat < input_n; ++feat) {
            // Use indices to update gradInput
            int index = static_cast<int>(indices_data[feat]);
            gradInput_data[index * input_n + feat] += gradOutput_data[feat];
        }
    }
}