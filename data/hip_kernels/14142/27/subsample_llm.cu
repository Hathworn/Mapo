#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // select input/output plane
    float *output_plane = output + o * output_w * output_h;
    float *input_plane = input + i * input_w * input_h;

    // Get the good mask for (k,i) (k out, i in)
    float the_weight = weight[k];
    
    // Initialize to the bias
    float the_bias = bias[k];

    // For all output pixels...
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            // Compute the mean of the input image...
            float *ptr_input = input_plane + yy * dH * input_w + xx * dW;
            float *ptr_output = output_plane + yy * output_w + xx;
            float sum = 0.0f;
            for (int ky = 0; ky < kH; ++ky) {
                for (int kx = 0; kx < kW; ++kx) {
                    sum += ptr_input[kx];
                }
                ptr_input += input_w; // next input line
            }
            // Update output
            *ptr_output = the_weight * sum + the_bias;
        }
    }
}