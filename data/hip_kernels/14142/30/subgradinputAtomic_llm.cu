#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate unique thread index
    int xx = blockIdx.x * blockDim.x + threadIdx.x;
    int yy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;
    
    // Ensure thread is within the bounds of the output
    if (xx < output_w && yy < output_h) {
        // Compute offsets based on thread/block ID
        int o = blockIdx.z;
        int i = o;

        // Select input/output plane
        gradOutput = gradOutput + o * output_w * output_h;
        gradInput = gradInput + i * input_w * input_h;

        // Get weight
        float the_weight = weight[blockIdx.z % input_n];

        // Compute pointer offsets and gradient contribution
        float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float z = *ptr_gradOutput * the_weight;

        // Apply atomicAdd in shared memory
        for (int ky = 0; ky < kH; ky++) {
            for (int kx = 0; kx < kW; kx++) {
                atomicAdd(&(ptr_gradInput[kx]), z);
            }
            ptr_gradInput += input_w;
        }
    }
}