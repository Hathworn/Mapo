#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_SpatialLogSoftMax_updateOutput_kernel(float *output, float *input, int classSize, int height, int width)
{
    int batchIndex = blockIdx.x;
    int index = threadIdx.x;

    // Use shared memory for faster memory access for each thread block
    extern __shared__ float shared_exp_sum[];

    while (index < height * width) {
        int y = index / width;
        int x = index % width;
        if (y >= height)
            break;

        // calculate input starting index in cuda layout (B x H x W x C)
        int inputStartIndex =
            (height * width * classSize) * batchIndex +
            (width * classSize) * y +
            (classSize) * x;

        // Use temporary variable to accumulate sum
        float sum = 0;
        for (int i = 0; i < classSize; i++) {
            sum += __expf(input[inputStartIndex + i]);
        }
        // Store sum in shared memory
        shared_exp_sum[threadIdx.x] = 1.0f / sum;
        __syncthreads();

        for (int i = 0; i < classSize; i++) {
            // calculate output index in torch layout (B x C x H x W)
            int outputIndex =
                (classSize * height * width) * batchIndex +
                (height * width) * i +
                (width) * y +
                x;
            output[outputIndex] = logf(shared_exp_sum[threadIdx.x] * __expf(input[inputStartIndex + i]));
        }
        index += blockDim.x;
    }
}