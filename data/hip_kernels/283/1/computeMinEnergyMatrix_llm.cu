#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeMinEnergyMatrix(float *energy, float *min_energy, int height, int width) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int pos = bid * blockDim.x + tid;

    extern __shared__ float shared_row_energy[];

    if (pos < width) {
        // Load initial row into shared memory, coalesced access
        for (int i = tid; i < width; i += blockDim.x) {
            shared_row_energy[i] = energy[i];
            min_energy[i] = energy[i];
        }
    } else {
        return;
    }
    __syncthreads();

    float temp[4];
    for (int i = 1; i < height; i++) {
        int k = 0;
        for (int j = tid; j < width; j += blockDim.x) {
            // Reduce divergence and unnecessary calculations
            float l = (j == 0) ? FLT_MAX : shared_row_energy[j - 1];
            float m = shared_row_energy[j];
            float r = (j == width - 1) ? FLT_MAX : shared_row_energy[j + 1];

            float minimum = energy[i * width + j] + min(l, min(m, r));
            temp[k++] = minimum;
        }
        __syncthreads();
        k = 0;
        for (int j = tid; j < width; j += blockDim.x) {
            shared_row_energy[j] = temp[k];
            min_energy[i * width + j] = temp[k++];
        }
        __syncthreads();  // Ensure all threads finish processing before next row
    }
}