#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize Matrix Multiplication kernel
__global__ void MatrixMul(int* a, int* b, int* c, int n) {
    // Calculate unique global row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int temp_sum = 0;

    // Boundary guard
    if (row < n && col < n) {
        // Loop unrolling for performance improvement
        for (int k = 0; k < n; k += 4) {
            temp_sum += a[row * n + k] * b[k * n + col];
            if (k+1 < n) temp_sum += a[row * n + (k+1)] * b[(k+1) * n + col];
            if (k+2 < n) temp_sum += a[row * n + (k+2)] * b[(k+2) * n + col];
            if (k+3 < n) temp_sum += a[row * n + (k+3)] * b[(k+3) * n + col];
        }
        c[row * n + col] = temp_sum;
    }
}