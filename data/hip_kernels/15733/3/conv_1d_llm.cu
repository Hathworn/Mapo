#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv_1d(int* a, int* b, int* c, int n, int m) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate the radius of the mask (mid point)
    int r = m / 2;

    // Initialize a temporary variable for accumulation
    int temp = 0;

    // Use shared memory for the mask to improve memory access speed
    __shared__ int shared_b[32]; // Assuming maximum mask size is 32
    if (threadIdx.x < m) {
        shared_b[threadIdx.x] = b[threadIdx.x];
    }
    __syncthreads(); // Ensure mask is fully loaded

    // Calculate the convolution only for valid indices
    for (int j = 0; j < m; j++) {
        int index = id + j - r;
        if (index >= 0 && index < n) {
            temp += a[index] * shared_b[j];
        }
    }

    // Assign result to the output array
    if (id < n) { // Ensure only valid index is assigned
        c[id] = temp;
    }
}