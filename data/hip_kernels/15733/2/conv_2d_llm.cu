#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv_2d(int* Mat, int* res, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= n || col >= n) return; // Early exit for out-of-bound threads

    int start_r = row - MASK_OFFSET;
    int start_c = col - MASK_OFFSET;

    int temp = 0;

    #pragma unroll // Unroll the loop for better performance
    for (int i = 0; i < MASK_LEN; i++) {
        #pragma unroll
        for (int j = 0; j < MASK_LEN; j++) {
            if ((start_r + i >= 0) && (start_r + i < n) && 
                (start_c + j >= 0) && (start_c + j < n)) {
                temp += Mat[(start_r + i) * n + (start_c + j)] * mask[i * MASK_LEN + j];
            }
        }
    }

    res[row * n + col] = temp;
}