#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv_1d(int* a, int* c, int n) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure threads only compute valid output elements
    if (id < n) {
        int r = MASK_LEN / 2;
        int start = id - r;
        int temp = 0;
        
        // Use shared memory for mask
        __shared__ int sharedMask[MASK_LEN];
        if (threadIdx.x < MASK_LEN) {
            sharedMask[threadIdx.x] = mask[threadIdx.x];
        }
        __syncthreads();

        // Calculate convolution with bounds check
        for (int j = 0; j < MASK_LEN; j++) {
            if ((start + j >= 0) && (start + j < n)) {
                temp += a[start + j] * sharedMask[j];
            }
        }
        c[id] = temp;
    }
}