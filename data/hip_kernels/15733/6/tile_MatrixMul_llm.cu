#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tile_MatrixMul(int* a, int* b, int* c, int n, int tile_size) {
    // Allocate shared memory statically
    __shared__ int A[Shared_Mem_Size];
    __shared__ int B[Shared_Mem_Size];

    // Calculate thread indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Calculate global row and column positions for this thread
    int row = by * tile_size + ty;
    int col = bx * tile_size + tx;

    // Intermediate sum for element being written
    int temp_val = 0;

    // Sweep tiles over the entire matrix
    for (int i = 0; i < (n + tile_size - 1) / tile_size; i++) {
        // Load one element into shared memory
        if (row < n && (i * tile_size + tx) < n) {
            A[(ty * tile_size) + tx] = a[row * n + (i * tile_size + tx)];
        } else {
            A[(ty * tile_size) + tx] = 0;
        }

        if (col < n && (i * tile_size + ty) < n) {
            B[(ty * tile_size) + tx] = b[(i * tile_size * n + ty * n) + col];
        } else {
            B[(ty * tile_size) + tx] = 0;
        }

        // Ensure all threads have loaded their data
        __syncthreads();

        // Calculate all temp values for this tile
        for (int j = 0; j < tile_size; j++) {
            temp_val += A[(ty * tile_size) + j] * B[(j * tile_size) + tx];
        }

        // Ensure all threads complete the computation before overwriting shared memory
        __syncthreads();
    }

    // Writeback to global memory
    if (row < n && col < n) {
        c[(row * n) + col] = temp_val;
    }
}