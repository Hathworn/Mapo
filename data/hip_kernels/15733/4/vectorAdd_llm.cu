#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(int* __restrict__ a, int* __restrict__ b, int* __restrict__ c, int n) {
    // Calculate global thread ID (tid)
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    // Use shared memory to minimize global memory access
    __shared__ int sharedA[BLOCK_SIZE];
    __shared__ int sharedB[BLOCK_SIZE];

    // Load data into shared memory
    if (tid < n) {
        sharedA[threadIdx.x] = a[tid];
        sharedB[threadIdx.x] = b[tid];
    }
    __syncthreads();

    // Vector boundary guard
    if (tid < n) {
        // Each thread adds a single element
        c[tid] = sharedA[threadIdx.x] + sharedB[threadIdx.x];
    }
}
```
Note: `BLOCK_SIZE` should be defined elsewhere to match the `blockDim.x`.