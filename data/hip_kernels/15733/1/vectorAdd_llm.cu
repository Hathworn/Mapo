#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(int* a, int* b, int* c, int n) {
    // Calculate global thread ID (tid)
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    // Optimize boundary guard and loop for efficiency
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        // Each thread adds multiple elements within its range
        c[i] = a[i] + b[i];
    }
}