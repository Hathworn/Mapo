#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double calculateValueOfWeight(double parameter, int sign, double alpha)
{
    return (parameter * sign * alpha);
}

__global__ void updateWeights(double* weights, double* parameters, double* otherp, int sign, double alpha)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure index is within bounds
    if (index < blockDim.x * gridDim.x) 
    {
        weights[index] += calculateValueOfWeight(parameters[index], sign, alpha);
    }
}