#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA kernel. Each thread takes care of one element of c

float* internalBuffer;
int nBuf;

__global__ void vecAdd(float *a, float *b, float *c, int n)
{
    // Use shared memory to improve memory access efficiency
    extern __shared__ float shared_mem[];

    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load data into shared memory
    if (id < n) {
        shared_mem[threadIdx.x] = a[id];
        __syncthreads(); // Ensure all threads have loaded their data
        shared_mem[threadIdx.x] += b[id];
        __syncthreads(); // Ensure all additions are done
        c[id] = shared_mem[threadIdx.x]; // Write result back to global memory
    }
}