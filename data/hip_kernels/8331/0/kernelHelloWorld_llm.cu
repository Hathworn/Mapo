#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelHelloWorld() {
    // Print from a single thread to avoid overloading stdout
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Hello World!\n");
    }
}