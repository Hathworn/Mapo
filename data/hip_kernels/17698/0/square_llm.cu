#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(int *array, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Use full threading hierarchy to generate unique thread id
    if (tid < n) {
        array[tid] *= array[tid]; // Simplify expression
    }
}