#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use blockIdx and threadIdx to parallelize for better performance
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        *accuracy /= N;
    }
}