#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fillSpaceTSteps(int N, int T, float c, float dt, float dd, float *waveSpace, float *waveSpaceTMin1, float *waveSpaceTMin2)
{
    // Calculate the 2D grid position
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Shared memory for tile, assuming block size is small enough to fit
    extern __shared__ float sWaveSpaceTMin1[];

    // Load data to shared memory; boundaries will load incomplete tiles
    if (i < N && j < N) {
        int idxLocal = threadIdx.y * blockDim.x + threadIdx.x;
        sWaveSpaceTMin1[idxLocal] = waveSpaceTMin1[N * i + j];
    }
    __syncthreads();

    // Perform calculation if within bounds
    if (i < N && j < N) {
        float cSquaredDtDd = (c * c) * (dt / dd * dt / dd);

        float neighborSum = ((i + 1 < N) ? waveSpaceTMin1[N * (i + 1) + j] : 0.0f)
                         + ((i - 1 >= 0) ? waveSpaceTMin1[N * (i - 1) + j] : 0.0f)
                         + ((j - 1 >= 0) ? waveSpaceTMin1[N * i + (j - 1)] : 0.0f)
                         + ((j + 1 < N) ? waveSpaceTMin1[N * i + (j + 1)] : 0.0f)
                         - 4 * sWaveSpaceTMin1[threadIdx.y * blockDim.x + threadIdx.x];

        waveSpace[N * i + j] = 2 * sWaveSpaceTMin1[threadIdx.y * blockDim.x + threadIdx.x] - waveSpaceTMin2[N * i + j] + cSquaredDtDd * neighborSum;
    }
}