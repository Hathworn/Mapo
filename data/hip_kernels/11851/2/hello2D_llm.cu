#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello2D()
{
    // Calculate unique global thread ID more efficiently
    int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    printf("I am thread (%d, %d) in block (%d, %d). Global thread ID = %d\n", threadIdx.y, threadIdx.x, blockIdx.y, blockIdx.x, tid);
}