#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(float *c, float* a, float *b, int values) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < values) {
        c[i] = a[i] + b[i]; // Perform element-wise addition for valid indices
    }
    // Removed unnecessary debugging print statement for cleaner code
}