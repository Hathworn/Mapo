#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* __restrict__ col_sum, int32_t* __restrict__ output, int32_t N) {
    int32_t idx = blockIdx.x * N + threadIdx.x;
    for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
        // Directly access memory with calculated index for each block
        output[idx] = -col_sum[i];
        idx += blockDim.x; // Increment index considering stride
    }
}