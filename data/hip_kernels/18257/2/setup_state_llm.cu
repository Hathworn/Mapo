#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize: Use block and thread indices for state setup to support multiple threads
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}