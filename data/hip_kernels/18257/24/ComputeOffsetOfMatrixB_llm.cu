#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    int32_t globalIdx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    int32_t totalThreads = gridDim.x * blockDim.x; // Total number of threads in the grid
    int32_t rowOffset = blockIdx.x * N; // Precompute row offset

    for (int32_t i = globalIdx; i < N; i += totalThreads) {
        output[rowOffset + i] = -row_sum[blockIdx.x];
    }
}