#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int n, long *a, long *b, long *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use warp reduction for smaller number of threads
    for (int stride = 1; stride < n; stride *= 2) {
        if (i % stride == 0 && i + stride < n) {
            a[i] += a[i + stride];
        }
        __syncthreads();
    }
    if (i < n) {
        sum[i] = a[i] + b[i];
        // Consider reducing or eliminating printf for efficiency
        printf("CUDA KERNEL ADD %ld + %ld = %ld \n", a[i], b[i], sum[i]);
    }
}