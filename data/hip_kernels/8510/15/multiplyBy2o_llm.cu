#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplyBy2o(int *size, const long *in, long *out) {
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for size to minimize global memory access
    __shared__ int shared_size;
    if (threadIdx.x == 0) {
        shared_size = *size;
    }
    __syncthreads();

    if (ix < shared_size) {
        out[ix] = in[ix] * 2;
    }
}