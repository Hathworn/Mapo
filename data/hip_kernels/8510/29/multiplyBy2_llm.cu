#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiplyBy2(int *size, int *in, int *out) {
    // Use a register variable for size to reduce memory access
    const int totalSize = *size;
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid unnecessary memory access by checking within the loop
    if (ix < totalSize) {
        out[ix] = in[ix] * 2;
    }
}