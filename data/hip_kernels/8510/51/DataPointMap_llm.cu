#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DataPointMap(int size, const double *inputX, const double *inputY, double *output, const double *inFreeArray, int length) {
    // Use long to avoid overflow in indexing for large sizes
    const long ix = blockDim.x * blockIdx.x + threadIdx.x;
    if (ix < size) {
        // Precompute offsets for input and output arrays
        const long offset = ix * length;
        const double *inArrayBody = inputX + offset;
        double *outArrayBody = output + offset;

        // Unroll loop for potential performance gain
        #pragma unroll
        for (long i = 0; i < length; i++) {
            outArrayBody[i] = inArrayBody[i] + inFreeArray[i];
        }
    }
}