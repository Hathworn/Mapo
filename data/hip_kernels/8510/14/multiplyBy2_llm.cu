#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiplyBy2(int size, const long *in, long *out) {
    // Use grid-stride loop for better workload distribution
    for (int ix = threadIdx.x + blockIdx.x * blockDim.x; ix < size; ix += blockDim.x * gridDim.x) {
        out[ix] = in[ix] * 2;
    }
}