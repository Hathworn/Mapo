#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void plusMinus(int *size, const double *base, const float *deviation, double *a, float *b) {
    // Load size into a register to minimize global memory access
    int n = *size;
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
    if (ix < n) {
        // Use a register to store intermediate results
        double baseVal = base[ix];
        float devVal = deviation[ix];
        a[ix] = baseVal - devVal;
        b[ix] = baseVal + devVal;
    }
}