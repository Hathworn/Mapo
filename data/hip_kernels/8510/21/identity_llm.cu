#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void identity(int *size, const int *input, int *output) {
    // Replace long with int for improved performance; GPU architecture favors int calculations
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if (ix < *size) {
        output[ix] = input[ix];
    }
}