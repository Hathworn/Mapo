#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorLength(int size, const double *x, const double *y, double *len) {
    // Calculate index outside loop to avoid redundant computation
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
    if (ix < size) {
        // Directly store the computation of x[ix] * x[ix] and y[ix] * y[ix] to reduce calculation steps
        double x_sq = x[ix] * x[ix];
        double y_sq = y[ix] * y[ix];
        len[ix] = sqrt(x_sq + y_sq);
    }
}