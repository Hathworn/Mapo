#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void plusMinus(int size, const double *base, const float *deviation, double *a, float *b) {
    const long ix = blockIdx.x * blockDim.x + threadIdx.x;  // Optimize thread index calculation
    if (ix < size) {
        double baseVal = base[ix];  // Cache memory access
        float deviationVal = deviation[ix];  // Cache memory access
        a[ix] = baseVal - deviationVal;  // Perform operation with cached values
        b[ix] = baseVal + deviationVal;  // Perform operation with cached values
    }
}