#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiplyBy2_l(int *size, long *in, long *out) {
    // Use a register to avoid pointer dereferencing
    const long total_size = *size;
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
    
    if (ix < total_size) {
        out[ix] = in[ix] * 2;
    }
}