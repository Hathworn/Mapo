#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void load(int size, const long *in) {
    // Calculate the global thread ID
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure index is within bounds
    if (ix < size) {
        // Prefetching data into registers to reduce memory latency
        long data = in[ix];
        // Here, you can perform some operations on `data` if needed
        // Currently `data` is loaded into register but not used
    }
}
