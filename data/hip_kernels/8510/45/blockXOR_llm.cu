#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blockXOR(int size, const char *input, char *output, long key) {
    // Optimize memory access and reduce redundant calculations
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
    const long idx = ix * 8;
    if (idx < size) {
        long *out = reinterpret_cast<long*>(output);
        const long *in = reinterpret_cast<const long*>(input);
        out[ix] = in[ix] ^ key;
    }
}