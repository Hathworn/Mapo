#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blockXOR(int *size, const char *input, char *output, long *key) {
    // Calculate the global index
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
    // Improved memory access by reading key to a register
    long keyVal = *key;
    // Check bounds and perform XOR operation
    if (ix * 8 < *size) {
        ((long *)output)[ix] = ((const long *)input)[ix] ^ keyVal;
    }
}