#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculateIntermediates(int n, double *xs, int *cluster_index, int *intermediates0, double *intermediates1, double *intermediates2, int k, int d) {
    int start = blockIdx.x * (n / gridDim.x);  // Define block start using grid size
    int end = min(start + (n / gridDim.x), n); // Use min to determine block end

    // Return if no work
    if (end <= start) return;

    // Loop for every K
    for (int clust = threadIdx.y; clust < k; clust += blockDim.y) {
        int count = 0; // Initialize count outside to share between dim

        // Loop for every dimension (features)
        for (int dim = threadIdx.x; dim < d; dim += blockDim.x) {
            double sum1 = 0.0;
            double sum2 = 0.0;
            
            for (int z = start; z < end; z++) {
                if (cluster_index[z] == clust) {
                    if (dim == 0) { 
                        count++; // Increment count for first dimension
                    }
                    int idx = z * d + dim;
                    sum1 += xs[idx];
                    sum2 += xs[idx] * xs[idx];
                }
            }
            if (dim == 0) {
                intermediates0[blockIdx.x * k + clust] = count;  // Set count outside dim loop
            }
            int index = (blockIdx.x * k * d + clust * d + dim);
            intermediates1[index] = sum1;
            intermediates2[index] = sum2;
        }
    }
}