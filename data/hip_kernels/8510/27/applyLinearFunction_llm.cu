#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void applyLinearFunction(int *size, const short *x, short *y, short *a, short *b) {
    // Use local int variable to reduce global memory access
    const int size_local = *size;
    const short a_local = *a;
    const short b_local = *b;
    
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
    
    // Add bounds check to ensure valid memory access
    if (ix < size_local) {
        // Compute linear function using local variables
        y[ix] = a_local + b_local * x[ix];
    }
}