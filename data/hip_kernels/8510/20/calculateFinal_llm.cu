#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculateFinal(int n, int *intermediates0, double *intermediates1, double *intermediates2, int *s0, double *s1, double *s2, int k, int d){

    // Check if blockIdx.x is 0 to ensure only one block is active
    if (blockIdx.x > 0) return;

    // Cache some frequently used data in registers
    int thread_x = threadIdx.x;
    int thread_y = threadIdx.y;
    int blockDim_y = blockDim.y;
    int blockDim_x = blockDim.x;

    // Loop for every cluster
    for (int clust = thread_y; clust < k; clust += blockDim_y){
        // Loop for every dimension (features)
        for (int dim = thread_x; dim < d; dim += blockDim_x) {

            // Calculate S0 - Only sum counts where dim is 0
            if (dim == 0) {
                int clustIndex = clust;
                // Unroll loop to minimize index calculations
                for (int z = clustIndex; z < 450 * k; z += k) {
                    s0[clust] += intermediates0[z];
                }
            }

            // Calculate S1 and S2
            int start = clust * d + dim;
            int kd = k * d;
            double *s1cur = &intermediates1[start];
            double *s2cur = &intermediates2[start];
            double tempS1 = 0.0;
            double tempS2 = 0.0;

            // Unroll loop for faster memory access
            double *s1end = s1cur + 450 * kd;
            for (; s1cur < s1end; s1cur += kd, s2cur += kd)
            {
                tempS1 += *s1cur;
                tempS2 += *s2cur;
            }
            s1[start] = tempS1;
            s2[start] = tempS2;
        }
    }
}