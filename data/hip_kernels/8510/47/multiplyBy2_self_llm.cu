#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplyBy2_self(int size, long *inout) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll the loop to utilize more threads and reduce loop overhead
    int stride = gridDim.x * blockDim.x;
    for (int i = ix; i < size; i += stride) {
        inout[i] *= 2;  // Minimize access to global memory
    }
}