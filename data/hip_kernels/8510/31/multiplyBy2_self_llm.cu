#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplyBy2_self(int *size, int *in, int *out) {
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;

    // Unroll loop to improve performance
    if (ix < *size) {
        int value = in[ix] * 2;
        out[ix] = value;
        in[ix] = value;
    }
}