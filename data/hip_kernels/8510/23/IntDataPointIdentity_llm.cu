#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void IntDataPointIdentity(int *size, const int *inputX, const int *inputY, int *outputX, int *outputY, int *length) {
    // Calculate global index
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
    
    // Check within bounds
    if (ix < *size) {
        // Use shared memory for input copy to optimize memory access
        extern __shared__ int sharedMemory[];
        const int *inArrayBody = &inputX[ix * *length];
        int *outArrayBody = &outputX[ix * *length];

        // Improved copy with shared memory usage
        for (long i = 0; i < *length; i++) {
            sharedMemory[i] = inArrayBody[i];
        }
        __syncthreads();
        for (long i = 0; i < *length; i++) {
            outArrayBody[i] = sharedMemory[i];
        }

        // Direct copy of scalar value
        outputY[ix] = inputY[ix];
    }
}