#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void IntDataPointIdentity(int size, const int *inputX, const int *inputY, int *outputX, int *outputY, int length) {
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
    if (ix < size) {
        // Use shared memory for improved memory access performance
        extern __shared__ int sharedMemory[];

        // Pointer for the shared memory block for current thread
        int *sharedArray = &sharedMemory[threadIdx.x * length];

        // Prefetch from global to shared memory
        const int *inArrayBody = &inputX[ix * length];
        for (long i = 0; i < length; i++) {
            sharedArray[i] = inArrayBody[i];
        }

        // Transfer from shared memory to global memory
        int *outArrayBody = &outputX[ix * length];
        for (long i = 0; i < length; i++) {
            outArrayBody[i] = sharedArray[i];
        }

        // Direct copy of int scalar value
        outputY[ix] = inputY[ix];
    }
}