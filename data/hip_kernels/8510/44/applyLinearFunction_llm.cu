#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void applyLinearFunction(int size, const short *x, short *y, short a, short b) {
    // Use an unsigned int for faster integer arithmetic operations
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    // Prefetch input element to reduce memory latency
    if (ix < size) {
        short xVal = x[ix];
        y[ix] = a + b * xVal;
    }
}