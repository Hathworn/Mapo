#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void intArrayIdentity(int *size, const int *input, int *output, int *length) {
    // Calculate the global index
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;

    if (ix < *size) {
        // Use pointers to traverse input and output arrays
        const int* inArrayBody = input + ix * (*length);
        int* outArrayBody = output + ix * (*length);

        // Optimize loop by unrolling
        for (long i = 0; i < *length; i++) {
            outArrayBody[i] = inArrayBody[i];
        }
    }
}