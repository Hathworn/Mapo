#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MNKernel(int count, long *Md, long *Nd, long *Pd, int width) {
    // 2D thread ID
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds to prevent out-of-bounds memory access
    if (row < count && col < width) {
        long Pvalue = 0;

        // Compute the dot product for the row and column
#pragma unroll
        for (int k = 0; k < width; k++) {
            Pvalue += Md[row * width + k] * Nd[k * width + col];
        }

        // Write the computed value to the matrix Pd only if within bounds
        Pd[row * width + col] = Pvalue;
    }
}