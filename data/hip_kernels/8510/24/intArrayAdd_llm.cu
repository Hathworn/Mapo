#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void intArrayAdd(int *size, const int *input, int *output, const int *inFreeArray, int *length) {
    // Calculate global index
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
    const long stride = blockDim.x * gridDim.x; // Calculate stride for grid-stride loop

    // Use grid-stride loop for accessing array elements
    for (long index = ix; index < *size * *length; index += stride) {
        int arrayIndex = index / *length; // Determine the starting index for this thread in input/output arrays
        int elementIndex = index % *length; // Determine the specific element in the array

        // Add elements from input arrays
        output[arrayIndex * *length + elementIndex] = input[arrayIndex * *length + elementIndex] + inFreeArray[elementIndex];
    }
}