#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplyBy2_self(int size, long *inout) {
    // Use shared memory to improve memory access speed and reduce global memory access
    extern __shared__ long sharedData[];
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix < size) {
        // Load from global memory to shared memory
        sharedData[threadIdx.x] = inout[ix];
        __syncthreads();

        // Perform computation using shared memory
        sharedData[threadIdx.x] *= 2;

        // Synchronize before writing back to global memory
        __syncthreads();
        inout[ix] = sharedData[threadIdx.x];
    }
}