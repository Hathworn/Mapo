#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiplyBy2(int size, long *in, long *out) {
    // Calculate the global index
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for better access efficiency
    __shared__ long sharedMem[1024]; // Assuming blockDim.x <= 1024

    if (ix < size) {
        // Load data from global to shared memory
        sharedMem[threadIdx.x] = in[ix];

        // Ensure all threads have loaded their data
        __syncthreads();

        // Perform operation using shared memory
        sharedMem[threadIdx.x] *= 2;

        // Ensure all operations are complete
        __syncthreads();

        // Write back to global memory
        out[ix] = sharedMem[threadIdx.x];
    }
}