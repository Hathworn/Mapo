#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void intArrayAdd(int size, const int *input, int *output, const int *inFreeArray, int length) {
    // Calculate global thread index
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;

    // Use shared memory for inFreeArray to reduce access latency
    __shared__ int sharedFreeArray[1024]; // Adjust size as needed

    // Load inFreeArray into shared memory (only by the first warp to avoid excessive shared memory use)
    if (threadIdx.x < length) {
        sharedFreeArray[threadIdx.x] = inFreeArray[threadIdx.x];
    }
    __syncthreads();

    if (ix < size) {
        // Pointers to the current segment of input and output arrays
        const int *inArrayBody = &input[ix * length];
        int *outArrayBody = &output[ix * length];

        // Loop over the length and perform addition
        for (long i = 0; i < length; i++) {
            outArrayBody[i] = inArrayBody[i] + sharedFreeArray[i];
        }
    }
}