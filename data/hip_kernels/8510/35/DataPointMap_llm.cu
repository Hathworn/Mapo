#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DataPointMap(int *size, const double *inputX, const double *inputY, double *output, const double *inFreeArray, int *length) {
    const long ix = threadIdx.x + blockIdx.x * (long)blockDim.x;
    if (ix < *size) {
        const double *inArrayBody = &inputX[ix * *length];
        double *outArrayBody = &output[ix * *length];

        // Unroll the loop for better performance
        for (long i = 0; i < *length; i += 4) {
            if (i + 0 < *length) outArrayBody[i + 0] = inArrayBody[i + 0] + inFreeArray[i + 0];
            if (i + 1 < *length) outArrayBody[i + 1] = inArrayBody[i + 1] + inFreeArray[i + 1];
            if (i + 2 < *length) outArrayBody[i + 2] = inArrayBody[i + 2] + inFreeArray[i + 2];
            if (i + 3 < *length) outArrayBody[i + 3] = inArrayBody[i + 3] + inFreeArray[i + 3];
        }
    }
}