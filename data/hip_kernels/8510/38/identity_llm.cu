#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void identity(int size, long *in, long *out) {
    // Compute global thread index
    const long ix = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if within bounds and copy data
    if (ix < size) {
        out[ix] = in[ix];
    }
}