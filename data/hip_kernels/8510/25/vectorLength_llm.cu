#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorLength(int *size, const double *x, const double *y, double *len) {
    const long ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix < *size) {
        // Pre-compute squares to avoid redundancy
        const double x_val = x[ix];
        const double y_val = y[ix];
        len[ix] = sqrt(x_val * x_val + y_val * y_val);
    }
}