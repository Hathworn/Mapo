#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void arrayTest(int n, long *factor, long *arr, long *result, int *const_arr1, long *const_arr2)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit if the thread index is out of bounds
    if (i >= n) return;

    int idx = i * 3;

    // Load factor[i] once to optimize memory access
    long factor_i = factor[i]; 

    // Unroll loop for better performance
    result[idx] = arr[idx] * factor_i;
    result[idx + 1] = arr[idx + 1] * factor_i;
    result[idx + 2] = arr[idx + 2] * factor_i;
}