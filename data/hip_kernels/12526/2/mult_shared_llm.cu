#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_DIM BLOCK_X

__global__ void mult_shared(int *A, int *B, int *result, int n) {
    const int col = blockIdx.x * TILE_DIM + threadIdx.x;
    const int row = blockIdx.y * TILE_DIM + threadIdx.y;

    __shared__ int tileA[TILE_DIM][TILE_DIM];
    __shared__ int tileB[TILE_DIM][TILE_DIM];

    int c = 0;

    for (int k = 0; k < n; k += TILE_DIM) {
        // Load shared memory tiles conditionally
        if (k + threadIdx.y < n && col < n)
            tileA[threadIdx.y][threadIdx.x] = A[col * n + k + threadIdx.y];
        else
            tileA[threadIdx.y][threadIdx.x] = 0;

        if (k + threadIdx.x < n && row < n)
            tileB[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y) * n + row];
        else
            tileB[threadIdx.y][threadIdx.x] = 0;

        __syncthreads(); // Synchronize all threads before computation

        // Compute partial results using shared tiles
        for (int kk = 0; kk < TILE_DIM; kk++) {
            c += tileA[threadIdx.y][kk] * tileB[kk][threadIdx.x];
        }

        __syncthreads(); // Synchronize all threads before loading the next tile
    }

    if (col < n && row < n)
        result[col * n + row] = c;
}