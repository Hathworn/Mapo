#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_global(int *A, int *B, int *result, int n)
{
    // Cache thread and block indices
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if current thread is within matrix bounds
    if (col < n && row < n)
    {
        int sum = 0;
        // Use a single loop to accumulate the result
        for (int k = 0; k < n; ++k)
        {
            sum += A[row * n + k] * B[k * n + col];
        }
        result[row * n + col] = sum; // Write the computed result outside the loop
    }
}