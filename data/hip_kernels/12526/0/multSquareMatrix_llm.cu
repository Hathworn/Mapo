#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 32

__global__ void multSquareMatrix(int *A, int *B, int *result, int n)
{
    __shared__ int shared_A[N][N];
    __shared__ int shared_B[N][N];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;
    int sum = 0;
    
    for (int m = 0; m < (n + N - 1) / N; m++) {
        // Load data into shared memory
        if (row < n && (m * N + tx) < n) {
            shared_A[ty][tx] = A[row * n + m * N + tx];
        } else {
            shared_A[ty][tx] = 0;
        }

        if (col < n && (m * N + ty) < n) {
            shared_B[ty][tx] = B[(m * N + ty) * n + col];
        } else {
            shared_B[ty][tx] = 0;
        }
        
        __syncthreads();  // Synchronize threads

        for (int k = 0; k < N; k++) {
            sum += shared_A[ty][k] * shared_B[k][tx];
        }
        
        __syncthreads();  // Synchronize again for loading new blocks
    }
    
    if (row < n && col < n) {
        result[row * n + col] = sum;
    }
}