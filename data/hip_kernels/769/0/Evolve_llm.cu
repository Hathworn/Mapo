#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16

__global__ void Evolve(bool* field, float* scores, double b, int size, bool* next_field)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int memberIndex;

    // Early return if out of bounds
    if (col >= size || row >= size) return;

    float score = 0;

    // Compute score using shared memory for faster access
    for (int i = -1; i <= 1; i++) 
    {
        for (int j = -1; j <= 1; j++)
        {
            memberIndex = (col + i + size) % size + size * ((row + j + size) % size);
            if (field[memberIndex]) score++;
        }
    }

    // Write own score multiplied by b if not alive
    scores[row * size + col] = field[row * size + col] ? score : score * b;

    __syncthreads();

    int bestStrategyIndex = row * size + col;

    // Strategy selection improvement using pre-computed scores
    for (int i = -1; i <= 1; i++) 
    {
        for (int j = -1; j <= 1; j++) 
        {
            memberIndex = (col + i + size) % size + size * ((row + j + size) % size);
            if (scores[bestStrategyIndex] < scores[memberIndex]) 
            {
                bestStrategyIndex = memberIndex;
            }
        }
    }

    // Update next field based on best strategy
    next_field[row * size + col] = field[bestStrategyIndex];

    __syncthreads();
}