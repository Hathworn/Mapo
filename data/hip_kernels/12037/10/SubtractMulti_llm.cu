#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_THREADS_X 16
#define MAX_THREADS_Y 16

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Optimize by pre-computing thread offset
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = threadIdx.z;

    int sz = height * pitch;
    int p = z * sz + y * pitch + x;

    // Combine if-check with computation to reduce divergence
    if (x < width && y < height)
    {
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
}