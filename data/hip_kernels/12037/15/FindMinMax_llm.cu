#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * blockDim.x + tx;
    const int y = blockIdx.y * 16;  // Optimized block size for stride access.
    int p = y * pitch + x;
    float val = FLT_MAX;  // Initialize to handle edge cases.
    
    // Load data into shared memory and handle out of bounds in unified manner.
    if (x < width) {
        val = d_Data[p];
    }
    minvals[tx] = val;
    maxvals[tx] = val;

    // Loop unrolled to reduce loop overhead.
    #pragma unroll
    for (int ty = 1; ty < 16; ty++) {
        p += pitch;
        if (x < width) {
            val = d_Data[p];
            if (val < minvals[tx]) minvals[tx] = val;
            if (val > maxvals[tx]) maxvals[tx] = val;
        }
    }
    __syncthreads();

    // Reduction to find minimum and maximum using binary reduction.
    for (int d = blockDim.x >> 1; d > 0; d >>= 1) {
        if (tx < d) {
            if (minvals[tx + d] < minvals[tx]) minvals[tx] = minvals[tx + d];
            if (maxvals[tx + d] > maxvals[tx]) maxvals[tx] = maxvals[tx + d];
        }
        __syncthreads();
    }

    // Store result back to global memory.
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}