#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputePositions(float *g_Data1, float *g_Data2, float *g_Data3, int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
    int i = blockIdx.x * POSBLK_SIZE + threadIdx.x;
    if (i >= numPts)
        return;

    int p = d_Ptrs[i];

    // Optimize data access by using registers instead of array
    float val0 = g_Data2[p];
    float dx = 0.5f * (g_Data2[p + 1] - g_Data2[p - 1]);
    float dxx = 2.0f * val0 - g_Data2[p - 1] - g_Data2[p + 1];
    float dy = 0.5f * (g_Data2[p + w] - g_Data2[p - w]);
    float dyy = 2.0f * val0 - g_Data2[p - w] - g_Data2[p + w];
    float ds = 0.5f * (g_Data1[p] - g_Data3[p]);
    float dss = 2.0f * val0 - g_Data3[p] - g_Data1[p];

    // Reuse common subexpressions to reduce redundant computations
    float val_p1_p1 = g_Data2[p + w + 1] + g_Data2[p - w - 1];
    float val_m1_p1 = g_Data2[p - w + 1] - g_Data2[p + w - 1];
    float dxy = 0.25f * (val_p1_p1 - val_m1_p1);

    float dxs = 0.25f * (g_Data3[p + 1] + g_Data1[p - 1] - g_Data1[p + 1] - g_Data3[p - 1]);
    float dys = 0.25f * (g_Data3[p + w] + g_Data1[p - w] - g_Data3[p - w] - g_Data1[p + w]);

    // Simplify expressions using common subexpressions
    float idxx = dyy * dss - dys * dys;
    float idxy = dys * dxs - dxy * dss;
    float idxs = dxy * dys - dyy * dxs;
    float idyy = dxx * dss - dxs * dxs;
    float idys = dxy * dxs - dxx * dys;
    float idss = dxx * dyy - dxy * dxy;
    float det = idxx * dxx + idxy * dxy + idxs * dxs;
    float idet = 1.0f / det;

    float pdx = idet * (idxx * dx + idxy * dy + idxs * ds);
    float pdy = idet * (idxy * dx + idyy * dy + idys * ds);
    float pds = idet * (idxs * dx + idys * dy + idss * ds);

    // Check if position adjustments are within valid range
    if (pdx < -0.5f || pdx > 0.5f || pdy < -0.5f || pdy > 0.5f || pds < -0.5f || pds > 0.5f) {
        pdx = __fdividef(dx, dxx);
        pdy = __fdividef(dy, dyy);
        pds = __fdividef(ds, dss);
    }

    float dval = 0.5f * (dx * pdx + dy * pdy + ds * pds);

    // Store results in global memory using efficient coalesced access
    d_Sift[i + 0 * maxPts] = (p % w) + pdx;
    d_Sift[i + 1 * maxPts] = (p / w) + pdy;
    d_Sift[i + 2 * maxPts] = d_ConstantA[0] * exp2f(pds * d_ConstantB[0]);
    d_Sift[i + 3 * maxPts] = val0 + dval;

    float tra = dxx + dyy;
    det = dxx * dyy - dxy * dxy;
    d_Sift[i + 4 * maxPts] = __fdividef(tra * tra, det);
}