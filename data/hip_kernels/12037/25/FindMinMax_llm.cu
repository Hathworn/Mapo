#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    
    int p = y * width + x;
    
    float val = (x < width) ? d_Data[p] : d_Data[p - x];
    
    minvals[tx] = val;
    maxvals[tx] = val;
    
    for (int ty = 1; ty < 16; ty++)
    {
        p += width;
        val = (x < width) ? d_Data[p] : val;
        if (val < minvals[tx]) minvals[tx] = val;
        if (val > maxvals[tx]) maxvals[tx] = val;
    }
    
    __syncthreads();
    
    for (int d = 1; d < blockDim.x; d <<= 1)
    {
        int mod = 2 * d;
        if (tx % mod == 0)
        {
            if (minvals[tx + d] < minvals[tx]) minvals[tx] = minvals[tx + d];
            if (maxvals[tx + d] > maxvals[tx]) maxvals[tx] = maxvals[tx + d];
        }
        __syncthreads();
    }
    
    if (tx == 0)
    {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}