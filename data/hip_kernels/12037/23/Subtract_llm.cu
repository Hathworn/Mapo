#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Use blockDim.x/y instead of hardcoded 16 for better flexibility
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int p = y * width + x; // Simplify to standard integer multiplication
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // Remove __syncthreads() as it is unnecessary in this context
}