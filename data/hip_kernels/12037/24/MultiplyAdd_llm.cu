#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Calculate the global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;

    // Check boundaries and perform computation
    if (x < width && y < height)
    {
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
}