#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    const int by = blockIdx.y;

    int numPts = 0;
    for (int y = by; y < h / 32; y += gridDim.y) { // Unroll loop by blockDim.y
        for (int x = bx * 16; x < w; x += gridDim.x * 16) { // Process by blockDim.x
            unsigned int val = minmax[y * w + x + tx];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}