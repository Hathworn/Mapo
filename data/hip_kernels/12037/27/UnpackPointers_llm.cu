#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;  // Use blockIdx.x for parallelization
    const int numThreads = blockDim.x * gridDim.x;
    int numPts = 0;

    // Loop over y and x in a parallelized manner
    for (int idx = bx * blockDim.x + tx; idx < (h/32)*w; idx += numThreads) {
        int y = idx / w;
        int x = idx % w;

        unsigned int val = minmax[y*w + x];
        if (val) {
            for (int k = 0; k < 32 && numPts < maxPts; k++) {
                if (val & 0x1) {
                    ptrs[16*numPts + tx] = (y*32 + k) * w + x;
                    numPts++;
                }
                val >>= 1;
            }
        }
    }
}