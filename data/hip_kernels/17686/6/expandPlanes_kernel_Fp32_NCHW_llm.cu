#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void expandPlanes_kernel_Fp32_NCHW(float* output, const uint64_t* masks, const float* values, int n) {
    // Block size of 256, optimized by reducing divergent branching and improving shared memory access.
    constexpr int kNumShmemElements = 4; // Matches the number of mask/val pairs

    __shared__ uint64_t shMasks[kNumShmemElements];
    __shared__ float shVals[kNumShmemElements];

    int index = threadIdx.x + blockDim.x * blockIdx.x;

    int planeIndex = index >> 6;

    if (planeIndex >= n) return;

    // Each warp of 64 threads collectively loads one mask/val pair to shared memory.
    if (threadIdx.x < 64) {
        if (threadIdx.x < kNumShmemElements) {
            shMasks[threadIdx.x] = masks[planeIndex + threadIdx.x];
            shVals[threadIdx.x] = values[planeIndex + threadIdx.x];
        }
    }
    __syncthreads();

    uint64_t mask = shMasks[planeIndex & (kNumShmemElements - 1)];
    float val = shVals[planeIndex & (kNumShmemElements - 1)];

    int sqIndex = index & 0x3F;
    output[index] = (mask & (1ull << sqIndex)) ? val : 0;
}