#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Using threadIdx.x to ensure only one thread modifies accuracy
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}