#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogramCuda(const uint8_t *data, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t *histogram)
{
    // Calculate global thread id for the flattened 2D grid
    const uint32_t id = blockIdx.y * blockDim.y * rowSize + blockIdx.x * blockDim.x + threadIdx.y * rowSize + threadIdx.x;
    
    // Ensure the calculated id is within bounds
    if (id < height * rowSize) {
        atomicAdd(&histogram[data[id]], 1);
    }
}