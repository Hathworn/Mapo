#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtractCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height) {
    // Use shared memory to reduce global memory access (if applicable)
    cPrconst uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        // Calculate linear index once to optimize accesses
        uint32_t idxIn1 = y * rowSizeIn1 + x;
        uint32_t idxIn2 = y * rowSizeIn2 + x;
        uint32_t idxOut = y * rowSizeOut + x;

        uint8_t val1 = in1[idxIn1];
        uint8_t val2 = in2[idxIn2];
        out[idxOut] = (val1 > val2) ? (val1 - val2) : 0;
    }
}