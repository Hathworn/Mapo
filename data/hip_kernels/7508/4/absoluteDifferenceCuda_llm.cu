#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void absoluteDifferenceCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
    // Calculate global thread index
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure index is within bounds
    if (x < width && y < height) {
        // Use single pointer arithmetic instead of additional variables
        uint32_t index = y * rowSizeIn1 + x;
        uint8_t val1 = in1[index];
        uint8_t val2 = in2[y * rowSizeIn2 + x];
        out[y * rowSizeOut + x] = (val1 > val2) ? (val1 - val2) : (val2 - val1);
    }
}