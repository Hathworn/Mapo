#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void minimumCuda(const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate flattened index to optimize memory access
    const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < width && idy < height) {
        const uint32_t index = idy * rowSizeOut + idx; // Use a single index calculation

        // Load, compare, and store the minimum value
        const uint8_t val1 = in1[idy * rowSizeIn1 + idx];
        const uint8_t val2 = in2[idy * rowSizeIn2 + idx];
        out[index] = min(val1, val2);
    }
}