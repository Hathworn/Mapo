#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void projectionProfileHorizontalCuda(const uint8_t *image, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t *projection) 
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        const uint8_t pixelValue = image[y * rowSize + x];
        // Use shared memory to reduce atomic operations
        __shared__ uint32_t partialSum[1024]; // Adjust size according to blockDim.x
        uint32_t tid = threadIdx.y * blockDim.x + threadIdx.x;
        partialSum[tid] = static_cast<uint32_t>(pixelValue);

        __syncthreads();

        // Reduce within block
        for (uint32_t stride = blockDim.y / 2; stride > 0; stride /= 2) {
            if (threadIdx.y < stride) {
                partialSum[tid] += partialSum[tid + stride * blockDim.x];
            }
            __syncthreads();
        }

        // One thread writes the result for each column to global memory
        if (threadIdx.y == 0) {
            atomicAdd(&projection[x], partialSum[threadIdx.x]);
        }
    }
}