#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void isAnyEqualCuda(const uint8_t * image, uint8_t * value, size_t valueCount, uint32_t width, uint32_t height, uint32_t * differenceCount)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure threads accessing image area within bounds
    if (x < width && y < height) {
        const uint32_t id = y * width + x;
        uint8_t pixelValue = image[id];

        // Unroll loop to reduce instruction overhead
        for (uint32_t i = 0; i < valueCount; i += 4) {
            if ((i < valueCount && pixelValue == value[i]) ||
                (i + 1 < valueCount && pixelValue == value[i + 1]) ||
                (i + 2 < valueCount && pixelValue == value[i + 2]) ||
                (i + 3 < valueCount && pixelValue == value[i + 3])) {
                atomicAdd(differenceCount, 1);
                return;  // Early return to avoid unnecessary further checks
            }
        }
    }
}