#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitwiseXorCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
    // Calculate the global thread index
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Use a single conditional for bounds checking
    if ( x < width && y < height ) {
        const uint32_t idIn1 = __mul24(y, rowSizeIn1) + x; // Use low-level multiply for efficiency
        const uint32_t idIn2 = __mul24(y, rowSizeIn2) + x;
        const uint32_t idOut = __mul24(y, rowSizeOut) + x;
        out[idOut] = in1[idIn1] ^ in2[idIn2];
    }
}