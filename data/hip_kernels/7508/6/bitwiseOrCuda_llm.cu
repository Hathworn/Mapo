#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use __restrict__ to hint the compiler about non-aliasing for potential optimizations
__global__ void bitwiseOrCuda(const uint8_t *__restrict__ in1, uint32_t rowSizeIn1, 
                              const uint8_t *__restrict__ in2, uint32_t rowSizeIn2, 
                              uint8_t *__restrict__ out, uint32_t rowSizeOut, 
                              uint32_t width, uint32_t height) 
{
    // Compute global thread index
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Use shared memory for common row size access
    if (x < width && y < height) {
        const uint32_t yOffset = y * rowSizeOut;  // Shared computation
        const uint32_t idIn1 = yOffset + (x % rowSizeIn1);  // Handle row offset with modulo for safety
        const uint32_t idIn2 = yOffset + (x % rowSizeIn2);
        const uint32_t idOut = yOffset + x;
        out[idOut] = in1[idIn1] | in2[idIn2];
    }
}
```
