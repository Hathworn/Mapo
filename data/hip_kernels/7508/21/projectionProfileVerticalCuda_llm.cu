#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void projectionProfileVerticalCuda(const uint8_t *image, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t *projection)
{
    // Use a single block dimension for x and loop over y for efficient access
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;

    if (x < width) {
        uint32_t sum = 0; // Use a register to store sum locally
        for (uint32_t y = 0; y < height; ++y) {
            const uint8_t *imageY = image + y * rowSize + x;
            sum += *imageY;
        }
        atomicAdd(&projection[blockIdx.x * blockDim.x + threadIdx.x], sum);
    }
}