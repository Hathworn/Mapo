#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setPixelCuda(uint8_t *in, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t *pointX, uint32_t *pointY, uint32_t pointSize, uint32_t value)
{
    // Use a single if condition to reduce branching
    const uint32_t idPoint = blockIdx.x * blockDim.x + threadIdx.x;
    if (idPoint < pointSize) {
        const uint32_t x = pointX[idPoint];
        const uint32_t y = pointY[idPoint];
        if ((x < width) & (y < height)) { // Use bitwise AND to optimize logic
            in[y * rowSize + x] = value;  // Optimize memory access pattern
        }
    }
}