#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void isEqualCuda(const uint8_t* in1, uint32_t rowSizeIn1, const uint8_t* in2, uint32_t rowSizeIn2, uint32_t width, uint32_t height, uint32_t* isEqual)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        // Optimize by combining declaration and assignment
        uint32_t partsEqual = (in1[y * rowSizeIn1 + x] == in2[y * rowSizeIn2 + x]) ? 1 : 0;
        atomicAnd(isEqual, partsEqual);
    }
}