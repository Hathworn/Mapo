#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flipCuda(const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height, bool horizontal, bool vertical)
{
    // Use shared memory for better memory access patterns
    __shared__ uint8_t tile[32][32];  // Assuming a block size of 32x32 for illustration

    const uint32_t inX = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t inY = blockDim.y * blockIdx.y + threadIdx.y;

    // Load data into shared memory
    if (inX < width && inY < height) {
        tile[threadIdx.y][threadIdx.x] = in[inY * rowSizeIn + inX];
    }
    __syncthreads();

    // Calculate flipped coordinates
    const uint32_t outX = horizontal ? (width - 1 - inX) : inX;
    const uint32_t outY = vertical ? (height - 1 - inY) : inY;

    // Write the flipped pixel value
    if (inX < width && inY < height) {
        out[outY * rowSizeOut + outX] = tile[threadIdx.y][threadIdx.x];
    }
}