#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void invertCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Optimized: Reading and writing data column-wise to improve memory coalescing
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        const uint32_t inputIndex = y * rowSizeIn + x;
        const uint32_t outputIndex = y * rowSizeOut + x;
        out[outputIndex] = ~in[inputIndex];
    }
}