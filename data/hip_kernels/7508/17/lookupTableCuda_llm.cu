#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lookupTableCuda(const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height, uint8_t * table)
{
    // Calculate global thread ID
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Check bounds and perform lookup
    if (x < width && y < height) {
        // Direct index calculations for increased readability
        uint32_t indexIn = y * rowSizeIn + x;
        uint32_t indexOut = y * rowSizeOut + x;
        out[indexOut] = table[in[indexIn]];
    }
}