#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void maximumCuda(const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate the unique thread index for the 2D grid
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Check if the thread is within image boundaries
    if (x < width && y < height) {
        // Compute the linear indices directly without intermediate variables
        uint32_t indexIn1 = y * rowSizeIn1 + x;
        uint32_t indexIn2 = y * rowSizeIn2 + x;
        uint32_t indexOut = y * rowSizeOut + x;

        // Perform the maximum operation
        out[indexOut] = (in1[indexIn1] > in2[indexIn2]) ? in1[indexIn1] : in2[indexIn2];
    }
}