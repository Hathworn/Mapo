#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rotateCuda(const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, float inXStart, float inYStart, uint32_t width, uint32_t height, float cosAngle, float sinAngle) {
    uint32_t outX = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t outY = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure the thread is processing a valid output pixel
    if (outX < width && outY < height) {
        // Calculate the input coordinates by inverse rotation
        const float exactInX = inXStart + cosAngle * outX + sinAngle * outY;
        const float exactInY = inYStart - sinAngle * outX + cosAngle * outY;

        const int32_t inX = static_cast<int32_t>(exactInX);
        const int32_t inY = static_cast<int32_t>(exactInY);

        uint8_t* outPix = out + outY * rowSizeOut + outX; // Pre-calculate output pixel pointer

        // Check boundary for valid input pixels
        if (inX < 0 || inX >= width - 1 || inY < 0 || inY >= height - 1) {
            *outPix = 0; // Set to 0 if out of bounds
        } else {
            const uint8_t* inPix = in + inY * rowSizeIn + inX; // Pre-calculate input pixel pointer
            const float probX = exactInX - inX;
            const float probY = exactInY - inY;
            
            // Simplified bilinear interpolation
            const float mean = inPix[0] * (1 - probX) * (1 - probY) +
                               inPix[1] * probX * (1 - probY) +
                               inPix[rowSizeIn] * (1 - probX) * probY +
                               inPix[rowSizeIn + 1] * probX * probY +
                               0.5f;
            *outPix = static_cast<uint8_t>(mean);
        }
    }
}