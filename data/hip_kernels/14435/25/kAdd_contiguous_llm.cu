#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread processes valid elements only
    if (idx < numEls) {
        dest[idx] = a[idx] + b[idx];
    }
}