#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void reduce(float *g_idata, float *g_odata, unsigned int n)
{
    extern __shared__ float sdata[];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x*2 + threadIdx.x;
    unsigned int gridSize = blockDim.x*2*gridDim.x;

    float mySum = 0;

    // Loop unrolling and boundary check in one step
    for (; i < n; i += gridSize) {
        mySum += g_idata[i];
        if (i + blockDim.x < n) 
            mySum += g_idata[i+blockDim.x];
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    if (blockDim.x >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockDim.x >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockDim.x >= 128) { if (tid <  64) { sdata[tid] += sdata[tid +  64]; } __syncthreads(); }

    // Unroll the loop for warp-level reduction
    if (tid < 32)
    {
        volatile float* smem = sdata;
        if (blockDim.x >=  64) smem[tid] += smem[tid + 32];
        if (blockDim.x >=  32) smem[tid] += smem[tid + 16];
        if (blockDim.x >=  16) smem[tid] += smem[tid +  8];
        if (blockDim.x >=   8) smem[tid] += smem[tid +  4];
        if (blockDim.x >=   4) smem[tid] += smem[tid +  2];
        if (blockDim.x >=   2) smem[tid] += smem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}