#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void add(int n, float *a, float *b, float *sum)
{
    // Use a shared memory cache to optimize memory access
    extern __shared__ float shared_a[];
    extern __shared__ float shared_b[];

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        // Load data into shared memory
        shared_a[threadIdx.x] = a[i];
        shared_b[threadIdx.x] = b[i];

        __syncthreads();

        // Use shared memory for computation
        sum[i] = shared_a[threadIdx.x] + shared_b[threadIdx.x];
    }
}