#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void invertVectorElements(float* vector, int n)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure threads operate within bounds
    if (i < n)
    {
        vector[i] = 1.0f / vector[i];
    }
}