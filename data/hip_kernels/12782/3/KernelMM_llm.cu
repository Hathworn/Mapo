#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE SIZE

__global__ void KernelMM(int N, int M, int P, float *A, float *B, float *C) {

    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];

    int bx = blockIdx.x;  
    int by = blockIdx.y;
    int tx = threadIdx.x; 
    int ty = threadIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;

    float tmp = 0.0;
    
    // Optimize: Unroll loop for better performance
    for (int m = 0; m < P; m += TILE_SIZE) {
        if (row < N && (m + tx) < P)
            sA[ty][tx] = A[row * P + m + tx];     
        else
            sA[ty][tx] = 0.0;

        if (col < M && (m + ty) < P)
            sB[ty][tx] = B[(m + ty) * M + col];
        else
            sB[ty][tx] = 0.0;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) 
            tmp += sA[ty][k] * sB[k][tx];

        __syncthreads();
    }
    
    if (row < N && col < M)
        C[row * M + col] = tmp;
}