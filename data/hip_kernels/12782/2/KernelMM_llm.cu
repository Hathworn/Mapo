#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 32 // Define tile size as a macro

__global__ void KernelMM(int N, int M, int P, float *A, float *B, float *C) {

    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;

    float tmp = 0.0;
    
    // Loop tiling for shared memory use
    for (int m = 0; m < P; m += TILE_SIZE) {
        // Coalesced loading of A and B into shared memory
        if (row < N && m + tx < P) 
            sA[ty][tx] = A[row * P + m + tx];
        else 
            sA[ty][tx] = 0.0;
        
        if (col < M && m + ty < P) 
            sB[ty][tx] = B[(m + ty) * M + col];
        else 
            sB[ty][tx] = 0.0;

        __syncthreads(); // Synchronize to ensure all data is loaded

        // Perform the multiplication for the partial tile
        for (int k = 0; k < TILE_SIZE; k++) {
            tmp += sA[ty][k] * sB[k][tx];
        }
        
        __syncthreads(); // Synchronize before loading new tile
    }
    
    if (row < N && col < M) // Ensure within bounds before storing the result
        C[row * M + col] = tmp;
}