#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 32 // Define tile size for shared memory usage

__global__ void KernelMM(int N, int M, int P, float *A, float *B, float *C) {

    __shared__ float sA[TILE_SIZE][TILE_SIZE]; // Use tile size for shared memory
    __shared__ float sB[TILE_SIZE][TILE_SIZE];

    int bx = blockIdx.x;  int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    int row = by * TILE_SIZE + ty;
    int col = bx * TILE_SIZE + tx;

    float tmp = 0.0f; // Use float literal for clarity

    for (int m = 0; m < (P + TILE_SIZE - 1) / TILE_SIZE; m++) {
        // Load data into shared memory with boundary checks
        if (row < N && m * TILE_SIZE + tx < P)
            sA[ty][tx] = A[row * P + m * TILE_SIZE + tx];
        else
            sA[ty][tx] = 0.0f;

        if (col < M && m * TILE_SIZE + ty < P)
            sB[ty][tx] = B[(m * TILE_SIZE + ty) * M + col];
        else
            sB[ty][tx] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; k++)
            tmp += sA[ty][k] * sB[k][tx];

        __syncthreads();
    }

    if (row < N && col < M)
        C[row * M + col] = tmp;
}