#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
#define _USE_MATH_DEFINES

__global__ void calculateDotProductsAndReduceGPU(int *vec1, int *vec2, int *reduced, int numElements)
{
    extern __shared__ int sdata[];

    int tid = threadIdx.x;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int numThreads = blockDim.x;

    // Initialize shared memory only if index is within range.
    if (i < numElements) {
        sdata[tid] = vec1[i] * vec2[i];
        sdata[tid + numThreads] = vec1[i] * vec1[i];
        sdata[tid + (numThreads * 2)] = vec2[i] * vec2[i];
    } else {
        sdata[tid] = 0;
        sdata[tid + numThreads] = 0;
        sdata[tid + (numThreads * 2)] = 0;
    }
    __syncthreads();

    // Optimize reduction using sequential addressing with reversed loop.
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
            sdata[tid + numThreads] += sdata[tid + numThreads + s];
            sdata[tid + (numThreads * 2)] += sdata[(tid + (numThreads * 2)) + s];
        }
        __syncthreads();
    }

    // Use atomic adds to update the reduced arrays.
    if (tid == 0) {
        atomicAdd(&reduced[0], sdata[tid]);
        atomicAdd(&reduced[1], sdata[tid + numThreads]);
        atomicAdd(&reduced[2], sdata[tid + (numThreads * 2)]);
    }
}