#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// richu shaji abraham richursa
using namespace std;

__global__ void scatter(int *d_array, int *d_scanArray, int *d_predicateArrry, int *d_scatteredArray, int d_numberOfElements, int offset)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Check boundary condition to avoid unnecessary computation
    if (index >= d_numberOfElements) return;
    
    // Only handle elements where predicate is true
    if (d_predicateArrry[index] == 1)
    {
        // Directly assign scattered value
        d_scatteredArray[d_scanArray[index] - 1 + offset] = d_array[index];
    }
}