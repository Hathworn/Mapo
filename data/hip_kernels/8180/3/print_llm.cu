#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print(int *d_predicateArrry, int numberOfElements)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within range
    if (index < numberOfElements) {
        printf("index = %d value = %d\n", index, d_predicateArrry[index]);
    }
}