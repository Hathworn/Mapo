#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void hillisSteeleScanDevice(int *d_array, int numberOfElements, int *d_tmpArray, int moveIndex)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    // Check if the index is within bounds
    if (index >= numberOfElements)
    {
        return;
    }
    // Load data to temporary array
    d_tmpArray[index] = d_array[index];
    // Perform the scan operation if valid
    if (index >= moveIndex)
    {
        d_tmpArray[index] += d_array[index - moveIndex];
    }
}