#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_histogram(unsigned int* bins, const float* dIn, const int binNumber, const float lumMin, const float lumMax, const int size) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the current thread's index is within bounds
    if (i >= size) // Changed '>' to '>=' to accurately handle the condition
        return;

    // Precompute reciprocal to avoid division in the loop
    float invRange = 1.0f / (lumMax - lumMin);
    int bin = ((dIn[i] - lumMin) * invRange) * binNumber;

    // Ensure that bin index is within the valid range [0, binNumber-1]
    bin = max(0, min(bin, binNumber - 1)); // Avoid possible out-of-bounds access

    atomicAdd(&bins[bin], 1);
}