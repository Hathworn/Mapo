#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blelloch_no_padding(unsigned int* d_in_array, const size_t numBins)
/*

\Params:
* d_in_array - input array of histogram values in each bin. Gets converted
to cdf by the end of the function.
* numBins - number of bins in the histogram (Must be < 2*MAX_THREADS_PER_BLOCK)
*/
{

    int thid = threadIdx.x;
    int offset = numBins >> 1; // Precompute offset for shared memory

    extern __shared__ float temp_array[];

    // Load input into shared memory
    temp_array[thid] = d_in_array[thid];
    temp_array[thid + offset] = d_in_array[thid + offset];
    __syncthreads();

    int stride = 1;
    // Up Sweep
    for (int d = offset; d > 0; d >>= 1) {
        if (thid < d) {
            int neighbor = stride * (2 * thid + 1) - 1;
            int index = stride * (2 * thid + 2) - 1;

            temp_array[index] += temp_array[neighbor];
        }
        stride <<= 1;
        __syncthreads();
    }

    // Set last element to identity
    if (thid == 0) temp_array[numBins - 1] = 0;

    // Down Sweep
    for (int d = 1; d < numBins; d <<= 1) {
        stride >>= 1;
        __syncthreads();

        if (thid < d) {
            int neighbor = stride * (2 * thid + 1) - 1;
            int index = stride * (2 * thid + 2) - 1;

            float t = temp_array[neighbor];
            temp_array[neighbor] = temp_array[index];
            temp_array[index] += t;
        }
    }

    __syncthreads();

    // Write results to global memory
    d_in_array[thid] = temp_array[thid];
    d_in_array[thid + offset] = temp_array[thid + offset];

}