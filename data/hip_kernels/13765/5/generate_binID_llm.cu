#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_binID(const float* dIn, int* out, const int binNumber, const float lumMin, const float lumMax, const int size) {
    // Calculate global thread index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (i < size) {
        // Calculate the range outside the loop
        float range = lumMax - lumMin;

        // Compute bin index
        int bin = __fdividef(dIn[i] - lumMin, range) * binNumber; // Use fast divide

        // Assign bin to output
        out[i] = bin;
    }
}