#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_bins(unsigned int* bins, int* in_binID, int binNumber, const int size) {
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x;
    int nt = blockDim.x;

    // Use a smaller shared array to reduce resource usage
    __shared__ unsigned int temp[1024];
    temp[tid] = 0;
    __syncthreads();

    // Use contiguous threads to improve memory access patterns
    for (int x = tid; x < size; x += nt) {
        if (in_binID[x] == i) {
            atomicAdd(&temp[tid], 1); // Use atomic operation to prevent race conditions
        }
        if (in_binID[x] > i) {
            break;
        }
    }
    __syncthreads();

    // Use a single thread to aggregate results using atomic operation
    if (tid == 0) {
        unsigned int sum = 0;
        for (int x = 0; x < nt; x++) {
            sum += temp[x];
        }
        atomicAdd(&bins[i], sum); // Use atomic operation to ensure correct updates
    }
}