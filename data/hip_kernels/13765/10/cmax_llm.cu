#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cmax(float *d_in, float *max, int len)
{
    extern __shared__ float smax[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load elements into shared memory and perform initial reduction step
    if (i < len)
        smax[tid] = d_in[i];
    else
        smax[tid] = -FLT_MAX; // Handle out-of-bounds access

    __syncthreads();

    // Unrolling loop for parallel reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride && (i + stride) < len) {
            smax[tid] = fmaxf(smax[tid], smax[tid + stride]);
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        max[blockIdx.x] = smax[0];
    }
}