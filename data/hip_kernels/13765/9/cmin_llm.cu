#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cmin(float *d_in, float *min, int len)
{
    extern __shared__ float smin[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Combined initial value assignment with boundary check
    smin[tid] = (i < len) ? d_in[i] : FLT_MAX; 

    __syncthreads();

    // Loop unrolling and warp synchronisation optimization
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            smin[tid] = min(smin[tid], smin[tid + s]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        min[blockIdx.x] = smin[0];
    }
}