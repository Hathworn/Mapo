#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_debug_kernel()
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index

    // Add any additional code or computation needed here using idx, if required
}