#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_curand_init(hiprandState *state, int seed)
{
    // Compute the global thread ID
    int global_id = blockDim.x * blockIdx.x + threadIdx.x;

    // Initialize hiprand state only if within bounds
    if (global_id < N) // Assume N is the maximum number of states
    {
        hiprand_init(seed, global_id, 0, &state[global_id]);
    }
}