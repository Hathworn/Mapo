#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_curand_init(hiprandState *state, int seed)
{
// Each possible thread uses same seed, but different sequence number
// (as suggested by CURAND docs)
int global_id = blockDim.x*blockIdx.x + threadIdx.x;
hiprand_init(seed,global_id,0,&state[global_id]);
}