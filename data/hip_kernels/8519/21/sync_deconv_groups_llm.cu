#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use __launch_bounds__ to specify the maximum number of threads per block and registers per thread
__global__ __launch_bounds__(256, 4) void sync_deconv_groups() {
    // Optimize memory usage and performance with empty kernel (no operation necessary)
    __syncthreads(); // Synchronize threads
}