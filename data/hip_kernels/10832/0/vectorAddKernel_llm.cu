#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAddKernel(float *a, float *b, float *c, int n) {
    // Using blockIdx.x and blockDim.x for efficient indexing
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Proper boundary check
    if (idx < n) {
        // Perform the vector addition
        c[idx] = a[idx] + b[idx];
    }
}