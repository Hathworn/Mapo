#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sqr_mag_kernel(const float *data, float *result, int total)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate 1D index.
    int dataIndex = 2 * idx; // Compute index for data array.

    if (idx < total) { // Ensure within bounds.
        float real = data[dataIndex]; // Load real part.
        float imag = data[dataIndex + 1]; // Load imaginary part.
        result[dataIndex] = real * real + imag * imag; // Compute squared magnitude.
        result[dataIndex + 1] = 0; // Set imaginary part to zero.
    }
}