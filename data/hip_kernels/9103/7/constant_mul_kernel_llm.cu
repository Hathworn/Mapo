#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constant_mul_kernel(const float *data_l, float constant, float *result, int total)
{
    // Calculate the global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int data_idx = 2 * idx;

    // Ensure the index is within bounds
    if (idx < total) {
        // Perform multiplication using coalesced memory access
        result[data_idx] = data_l[data_idx] * constant;
        result[data_idx + 1] = data_l[data_idx + 1] * constant;
    }
}