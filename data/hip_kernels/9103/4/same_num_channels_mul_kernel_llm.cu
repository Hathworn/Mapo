#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void same_num_channels_mul_kernel(const float *data_l, const float *data_r, float *result, int total)
{
    int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);

    // Optimize calculation by checking condition prior to computations
    if (idx < total * 2) {
        float left_real = data_l[idx];
        float left_imaginary = data_l[idx + 1];
        float right_real = data_r[idx];
        float right_imaginary = data_r[idx + 1];

        // Store results in registers before writing to global memory
        float res_real = left_real * right_real - left_imaginary * right_imaginary;
        float res_imaginary = left_real * right_imaginary + left_imaginary * right_real;

        result[idx] = res_real;
        result[idx + 1] = res_imaginary;
    }
}