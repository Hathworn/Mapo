#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constant_add_kernel(const float *data_l, float constant, float *result, int total)
{
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x; 
    
    // Compute the effective input index
    int idx = 2 * tid;

    // Check if within limits and only access when necessary
    if (idx < 2 * total) {
        result[idx] = data_l[idx] + constant;
        result[idx + 1] = data_l[idx + 1];
    }
}