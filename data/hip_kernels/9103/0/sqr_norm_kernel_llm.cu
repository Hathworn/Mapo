#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sqr_norm_kernel(const float *in, float *block_res, int total)
{
    extern __shared__ float sdata[];
    int in_idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
    int i = threadIdx.x;
    unsigned ins = blockDim.x;

    // Load into shared memory
    sdata[i] = (in_idx < total * 2) ? (in[in_idx] * in[in_idx] + in[in_idx + 1] * in[in_idx + 1]) : 0;
    __syncthreads();

    // In-place reduction using shared memory
    for (unsigned outs = (ins + 1) / 2; ins > 1; ins = outs, outs = (outs + 1) / 2) {
        if (i + outs < ins)
            sdata[i] += sdata[i + outs];
        __syncthreads();
    }

    // Write result for this block to global memory
    if (i == 0)
        block_res[blockIdx.x] = sdata[0];
}