#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conj_kernel(const float *data, float *result, int total)
{
    // Calculate index in a more efficient way using '2 * threadIdx.x'
    int idx = 2 * threadIdx.x + 2 * blockIdx.x * blockDim.x;

    if (idx < 2 * total) {
        result[idx] = data[idx];              // Copy real part
        result[idx + 1] = -data[idx + 1];     // Negate imaginary part
    }
}