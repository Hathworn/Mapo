#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void same_num_channels_div_kernel(const float *data_l, const float *data_r, float *result, unsigned total)
{
    unsigned idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);

    // Check if index is within bounds
    if (idx < 2 * total) {
        float real_l = data_l[idx];
        float imag_l = data_l[idx + 1];
        float real_r = data_r[idx];
        float imag_r = data_r[idx + 1];

        float denominator = real_r * real_r + imag_r * imag_r;

        // Calculation of real part
        result[idx] = (real_l * real_r + imag_l * imag_r) / denominator;

        // Calculation of imaginary part
        result[idx + 1] = (imag_l * real_r - real_l * imag_r) / denominator;
    }
}