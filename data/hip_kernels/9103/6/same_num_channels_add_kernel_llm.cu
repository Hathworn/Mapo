#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void same_num_channels_add_kernel(const float *data_l, const float *data_r, float *result, int total)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = 2 * idx;
    
    // Each thread processes 2 elements to improve coalesced memory access
    if (idx < total) {
        result[offset] = data_l[offset] + data_r[offset];
        result[offset + 1] = data_l[offset + 1] + data_r[offset + 1];
    }
}