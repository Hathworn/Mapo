#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_hello() {
    // Each thread prints a message. Use a single thread to avoid redundant prints.
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Hello World from GPU!\n");
    }
}