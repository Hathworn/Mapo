#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index

    if (index < n) { // Ensure index is within bounds
        out[index] = a[index] + b[index];
    }
}