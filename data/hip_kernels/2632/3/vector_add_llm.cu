#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vector_add(float *out, float *a, float *b, int n) {
    // Use thread and block indices for parallel computation
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = index; i < n; i += stride) {
        out[i] = a[i] + b[i];
    }
}