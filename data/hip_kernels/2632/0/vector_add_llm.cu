#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vector_add(float *out, float *a, float *b, int n) {
    // Calculate the global index for the current thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds before performing operation
    if (index < n) {
        out[index] = a[index] + b[index];
    }
}