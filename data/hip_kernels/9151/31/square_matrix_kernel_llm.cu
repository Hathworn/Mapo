#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Use shared memory to reduce global memory accesses
    __shared__ float shared_feats[1024]; // Adjust size as needed based on occupancy and architecture

    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Prefetch data into shared memory if within bounds
    if (i < num_rows && j < num_cols) {
        shared_feats[tid] = feats[i * ldf + j];
        __syncthreads(); // Ensure all threads have loaded their data

        // Use shared memory for computation
        float f = shared_feats[tid];
        feats_sq[i * lds + j] = f * f;
    }
}