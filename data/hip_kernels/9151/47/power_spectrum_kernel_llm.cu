#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;

    // Use shared memory to reduce global memory accesses
    __shared__ float2 shared_Ar[CU1DBLOCK];
    if (thread_id < half_length) {
        shared_Ar[thread_id] = reinterpret_cast<float2 *>(Ar)[thread_id];
    }
    __syncthreads();

    for (int idx = thread_id; idx < half_length; idx += CU1DBLOCK) {
        // Ignore special case
        if (idx == 0) continue;

        // Use shared memory instead of global memory directly
        float2 val = shared_Ar[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret);
    }

    // Handle special case using first thread
    if (threadIdx.x == 0) {
        float real = Ar[0];
        // cufft puts this at the end, different than kaldi does with its own internal implementation
        float im = Ar[row_length];

        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}