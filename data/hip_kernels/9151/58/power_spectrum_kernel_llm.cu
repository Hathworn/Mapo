#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    const float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    int idx = thread_id;

    // Use shared memory for better memory access performance
    __shared__ float data_shared[CU1DBLOCK][2];

    while (idx < half_length) {
        if (idx != 0) { // skip special case
            data_shared[thread_id][0] = Ar[2 * idx];   // load real part
            data_shared[thread_id][1] = Ar[2 * idx + 1]; // load imaginary part
            __syncthreads();

            float ret = data_shared[thread_id][0] * data_shared[thread_id][0] +
                        data_shared[thread_id][1] * data_shared[thread_id][1];
            if (use_power) {
                Aw[idx] = ret;
            } else {
                Aw[idx] = sqrtf(ret);
            }
        }
        idx += CU1DBLOCK;
        __syncthreads();
    }

    // Handle special case
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        if (use_power) {
            Aw[0] = real * real;
            Aw[half_length] = im * im;
        } else {
            Aw[0] = fabs(real);
            Aw[half_length] = fabs(im);
        }
    }
}