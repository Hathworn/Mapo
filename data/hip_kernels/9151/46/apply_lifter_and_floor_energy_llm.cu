#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    float *feats = features + frame * ldf;

    // Apply lifter coefficients if cepstral_lifter is non-zero
    if (cepstral_lifter != 0.0f) {
        for (int c = thread_id; c < num_cols; c += blockDim.x) {
            feats[c] *= lifter_coeffs[c];  // Simplified lifting operation
        }
    }

    // Use energy only if conditions are met, thread 0 handles energy flooring
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        if (energy_floor > 0.0f) {
            float log_energy_floor = log(energy_floor);
            if (energy < log_energy_floor) {
                energy = log_energy_floor;
            }
        }
        feats[0] = energy;
    }
}