#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device_new_api(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    // Unroll loop for efficiency
    for (i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = fmaxf(val, largest);
    }
    for (i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp);
        sum += e;
        output[i * stride] = e;
    }
    float inv_sum = 1.0f / sum; // Precompute reciprocal
    for (i = 0; i < n; ++i) {
        output[i * stride] *= inv_sum;
    }
}

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    // Unroll loop for efficiency
    for (i = 0; i < n; ++i) {
        float val = input[i];
        largest = fmaxf(val, largest);
    }
    for (i = 0; i < n; ++i) {
        float e = expf((input[i] - largest) / temp);
        sum += e;
        output[i] = e;
    }
    float inv_sum = 1.0f / sum; // Precompute reciprocal
    for (i = 0; i < n; ++i) {
        output[i] *= inv_sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;
    int s = id % spatial;
    id = id / spatial;
    int g = id % groups;
    int b = id / groups;
    int goff = group_offset[g] * spatial;
    int boff = b * stride;
    // Use the optimized softmax function
    softmax_device_new_api(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}