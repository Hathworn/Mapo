#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    extern __shared__ float shared_memory[]; // Utilize shared memory
    float *e_vals = shared_memory;
    float sum = 0.0f;
    float largest = -INFINITY;

    // Find largest value in a single loop
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        float val = input[i];
        atomicMax((int*)&largest, __float_as_int(val)); // Compute largest in parallel
    }
    __syncthreads();

    // Compute exponentials and their sum
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        float e = expf((input[i] - largest) / temp);
        e_vals[i] = e; // Store exponentials in shared memory
        atomicAdd(&sum, e);
    }
    __syncthreads();

    // Normalize output values
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        output[i] = e_vals[i] / sum;
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if (b >= batch) return;
    softmax_device(n, input + b * offset, temp, output + b * offset);
}