#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    if (index < N) {  // Check if within bounds
        int f = (index / spatial) % filters;  // Calculate filter index
        x[index] = (x[index] - mean[f]) / (sqrtf(variance[f]) + .000001f);  // Normalize
    }
}