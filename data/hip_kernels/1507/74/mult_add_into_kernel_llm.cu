#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate a unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index is within bounds
    if(i < n){
        // Perform multiplication and addition
        c[i] += a[i] * b[i];
    }
}