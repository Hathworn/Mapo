#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void im2col_align_gpu_kernel(const int n, const float* data_im, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_col, const int bit_align)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop for better occupancy
    for (; index < n; index += blockDim.x * gridDim.x) {
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;

        // Pre-calculate some indices 
        float* data_col_ptr = &data_col[channel_out * bit_align + h_out * width_col + w_out];
        const float* data_im_ptr = data_im + (channel_in * height + h_in) * width + w_in;
        int pre_out_index = index % (width_col * height_col);

        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                float val = (h >= 0 && w >= 0 && h < height && w < width) ?
                            data_im_ptr[i * width + j] : 0;

                int out_index = (channel_out + i*ksize + j) * bit_align + pre_out_index;
                data_col[out_index] = val;

                // Move this move out of inner loop for better performance
                // data_col_ptr += bit_align;
            }
        }
    }
}