#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Optimize by using shared memory to reduce global memory access latency
    extern __shared__ float shared_X[];
    
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    if (i < N) {
        // Load data into shared memory
        shared_X[tid] = X[OFFX + i * INCX];
        __syncthreads();

        // Perform AXPY operation from shared memory
        Y[OFFY + i * INCY] += ALPHA * shared_X[tid];
    }
}