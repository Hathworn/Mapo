#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Pre-calculate constants outside the loop
    const float scale = 1.0F / (batch * spatial - 1);
    const int stride = filters * spatial;
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float var_sum = 0.0f;

    for (int j = 0; j < batch; ++j) {
        int base_index = j * stride + i * spatial;
        for (int k = 0; k < spatial; ++k) {
            float diff = x[base_index + k] - mean[i]; // Reuse subtracted value
            var_sum += diff * diff; // Avoid powf for squaring
        }
    }
    
    variance[i] = var_sum * scale;
}