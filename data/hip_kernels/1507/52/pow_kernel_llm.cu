#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Optimize the calculation of global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use an appropriate stride to process all elements
    int stride = blockDim.x * gridDim.x;
    
    for (int i = idx; i < N; i += stride) {
        Y[i * INCY] = powf(X[i * INCX], ALPHA);
    }
}