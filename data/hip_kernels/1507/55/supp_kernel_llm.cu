#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Flatten block index computation
    int stride = gridDim.x * blockDim.x; // Calculate stride for looping
    while (i < N) { // Use loop to cover larger dataset
        float value = X[i*INCX]; // Cache value to avoid repeated memory access
        if (value * value < ALPHA * ALPHA) X[i*INCX] = 0;
        i += stride; // Increment index by stride
    }
}