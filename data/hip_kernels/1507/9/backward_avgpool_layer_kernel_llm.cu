#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplify thread index calculation
    if (id >= n * c) return;

    int k = id % c;
    int b = id / c;

    int out_index = k + c * b; // Compute once, used inside loop
    int num_elements = w * h; // Calculate total elements in advance

    // Unroll loop for improved efficiency, assuming w*h is a multiple of 4
    for (int i = 0; i < num_elements; i += 4) {
        int in_index = i + num_elements * (k + b * c);
        float delta = out_delta[out_index] / num_elements;
        in_delta[in_index] += delta;
        if(i + 1 < num_elements) in_delta[in_index + 1] += delta;
        if(i + 2 < num_elements) in_delta[in_index + 2] += delta;
        if(i + 3 < num_elements) in_delta[in_index + 3] += delta;
    }
}