#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global index directly
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure all threads that are in bounds perform the operation
    if (i < N) {
        Y[i * INCY] *= X[i * INCX];
    }
}