#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process only elements within bounds
    if (i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);  // Use fabsf for float absolute value

        // Optimize branching logic
        if (abs_val < 1.0f) {
            error[i] = diff * diff;
            delta[i] = diff;
        } else {
            error[i] = 2.0f * abs_val - 1.0f;
            delta[i] = copysignf(1.0f, diff);  // Use copysignf to determine delta value
        }
    }
}