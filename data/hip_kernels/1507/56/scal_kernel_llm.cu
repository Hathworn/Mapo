#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate unique thread index for 1D block and grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride of block size to allow processing of larger data
    int stride = blockDim.x * gridDim.x;
    // Loop over the array in steps of block size
    for (; i < N; i += stride)
    {
        X[i * INCX] *= ALPHA;
    }
}