#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device_new_api(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    for (i = 0; i < n; ++i) {
        float val = input[i * stride]; // Change type to float for precision
        largest = max(val, largest);   // Use max function for clarity
    }
    for (i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp); // Optimize exp calculation
        sum += e;
        output[i * stride] = e;
    }
    for (i = 0; i < n; ++i) {
        output[i * stride] /= sum; // Normalize the output
    }
}

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    for (i = 0; i < n; ++i) {
        float val = input[i];
        largest = max(val, largest);
    }
    for (i = 0; i < n; ++i) {
        float e = expf((input[i] - largest) / temp);
        sum += e;
        output[i] = e;
    }
    for (i = 0; i < n; ++i) {
        output[i] /= sum;
    }
}

__global__ void softmax_kernel_new_api(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x; // Optimize id calculation
    if (id >= batch * groups) return;  // Early exit for out of bounds checks
    int b = id / groups;
    int g = id % groups;
    softmax_device_new_api(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}