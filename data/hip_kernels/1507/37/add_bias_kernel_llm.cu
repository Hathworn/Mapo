#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    // Cache biases in shared memory for faster access
    __shared__ float shared_biases[1024]; // Adjust size according to maximum filters

    int filter = blockIdx.y;
    if (threadIdx.x == 0) {
        shared_biases[filter] = biases[filter];
    }
    __syncthreads();

    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int batch = blockIdx.z;

    // Ensure valid offset before performing calculation
    if(offset < size) {
        output[(batch * n + filter) * size + offset] += shared_biases[filter];
    }
}