#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    __shared__ float sharedMean[1024]; // Use shared memory for partial results
    float scale = 1.F / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    sharedMean[threadIdx.x] = 0; // Initialize shared memory
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sharedMean[threadIdx.x] += x[index]; // Accumulate in shared memory
        }
    }
    mean[i] = sharedMean[threadIdx.x] * scale; // Use shared memory result
}