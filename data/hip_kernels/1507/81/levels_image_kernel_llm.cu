#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float3 hsv_to_rgb_kernel(float3 hsv) {
    float h = hsv.x, s = hsv.y, v = hsv.z;
    float r = v, g = v, b = v;
    if (s > 0) {  // Improved readability and reduced nesting
        int index = (int)floorf(h);
        float f = h - index;
        float p = v * (1 - s);
        float q = v * (1 - s * f);
        float t = v * (1 - s * (1 - f));

        if (index == 0) { r = v; g = t; b = p; }
        else if (index == 1) { r = q; g = v; b = p; }
        else if (index == 2) { r = p; g = v; b = t; }
        else if (index == 3) { r = p; g = q; b = v; }
        else if (index == 4) { r = t; g = p; b = v; }
        else { r = v; g = p; b = q; }
    }
    // Clamp values
    r = fmaxf(0, fminf(r, 1));
    g = fmaxf(0, fminf(g, 1));
    b = fmaxf(0, fminf(b, 1));
    return make_float3(r, g, b);
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb) {
    float r = rgb.x, g = rgb.y, b = rgb.z;
    float max = fmaxf(r, fmaxf(g, b)), min = fminf(r, fminf(g, b));
    float delta = max - min, h, s = (max == 0) ? 0 : delta / max, v = max;
    if (delta > 0) {
        if (r == max) { h = (g - b) / delta; }
        else if (g == max) { h = 2 + (b - r) / delta; }
        else { h = 4 + (r - g) / delta; }
        if (h < 0) h += 6;
    } else { h = -1; }
    return make_float3(h, s, v);
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift) {
    int size = batch * w * h;
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;

    int x = id % w, y = (id / w) % h, idx = id / (w * h);
    float3 rgb;
    float rshift = rand[0], gshift = rand[1], bshift = rand[2];
    float r0 = rand[8 * idx + 0], r1 = rand[8 * idx + 1], r2 = rand[8 * idx + 2], r3 = rand[8 * idx + 3];

    saturation = r0 * (saturation - 1) + 1;
    saturation = (r1 > 0.5f) ? 1.0f / saturation : saturation;
    exposure = r2 * (exposure - 1) + 1;
    exposure = (r3 > 0.5f) ? 1.0f / exposure : exposure;

    size_t offset = idx * h * w * 3;
    image += offset;
    rgb = make_float3(image[x + w * (y + h * 0)], image[x + w * (y + h * 1)], image[x + w * (y + h * 2)]);

    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else { shift = 0; }

    image[x + w * (y + h * 0)] = rgb.x * scale + translate + (rshift - 0.5f) * shift;
    image[x + w * (y + h * 1)] = rgb.y * scale + translate + (gshift - 0.5f) * shift;
    image[x + w * (y + h * 2)] = rgb.z * scale + translate + (bshift - 0.5f) * shift;
}