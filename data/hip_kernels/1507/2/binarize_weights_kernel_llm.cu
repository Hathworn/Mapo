#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    // Calculate the index for each thread
    int f = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (f >= n) return;

    float mean = 0.0f;
    float weight_sum = 0.0f;
    
    // Compute sum of absolute weights
    for (int i = 0; i < size; ++i) {
        weight_sum += fabs(weights[f * size + i]);
    }
    mean = weight_sum / size; // Calculate mean of absolute values

    // Binarize weights
    for (int i = 0; i < size; ++i) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    }
}