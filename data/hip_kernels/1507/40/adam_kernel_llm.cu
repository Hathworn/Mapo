#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation
    if (index >= N) return;

    float B1_t = 1.f - powf(B1, t); // Precompute (1 - B1^t) for efficiency
    float B2_t = 1.f - powf(B2, t); // Precompute (1 - B2^t) for efficiency
    float mhat = m[index] / B1_t;
    float vhat = v[index] / B2_t;

    x[index] += rate * mhat / (sqrtf(vhat) + eps); // Use compound assignment for clarity
}