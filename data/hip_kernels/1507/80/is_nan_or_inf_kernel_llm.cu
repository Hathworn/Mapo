#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void is_nan_or_inf_kernel(float *input, size_t size, int *pinned_return)
{
    // Calculate global index
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check within bounds
    if (index < size)
    {
        // Fetch the value
        float val = input[index];
        
        // Atomic operation to avoid race conditions
        if (isnan(val) || isinf(val))
        {
            atomicExch(pinned_return, 1);
        }
    }
}