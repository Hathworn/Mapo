#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop over all elements with the stride of total number of threads
    for (; i < N; i += blockDim.x * gridDim.x) {
        X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i * INCX]));
    }
}