#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use blockIdx and threadIdx to calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Use strided loops for better memory access patterns and occupancy
    for (; i < N; i += gridDim.x * gridDim.y * blockDim.x)
    {
        X[i * INCX] = ALPHA; // Write the constant ALPHA to the stride of INCX
    }
}