#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for softmax cross-entropy
__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use a stride loop to enhance performance, especially for large n
    for (; i < n; i += stride) {
        float t = truth[i];
        float p = pred[i];
        error[i] = (t) ? -log(p) : 0;
        delta[i] = t - p;
    }
}