#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    // Use flattened grid and block indices for optimal thread mapping
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check boundary condition before accessing global memory
    if (i < n && mask[i] == mask_num)
    {
        // Efficient memory write by reducing condition checks
        x[i] = mask_num;
    }
}