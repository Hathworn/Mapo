#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Abort if out of bounds
    if (id >= size) return;

    // Optimized index calculations
    int i = id % minw;
    int j = (id / minw) % minh;
    int k = (id / (minw * minh)) % minc;
    int b = id / (minw * minh * minc);

    // Streamlined index calculation
    int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
    int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));

    atomicAdd(&out[out_index], add[add_index]); // Use atomic for concurrent writes
}