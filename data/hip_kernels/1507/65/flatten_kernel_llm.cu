#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate unique index for the thread
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    
    // Boundary check
    if(i >= N) return;

    // Pre-compute indices to reduce redundant calculations
    int b = i / (spatial * layers);
    int rem = i % (spatial * layers);
    int in_c = rem / spatial;
    int in_s = rem % spatial;

    // Compute source and destination indices for flatten operation
    int i1 = b*layers*spatial + in_c*spatial + in_s;
    int i2 = b*layers*spatial + in_s*layers + in_c;

    // Copy data forward or backward into output based on the given flag
    if (forward) out[i2] = x[i1];
    else out[i1] = x[i2];
}