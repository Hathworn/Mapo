#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Use blockIdx.z for additional parallelism
    int i = (blockIdx.x + blockIdx.y*gridDim.x + blockIdx.z*gridDim.x*gridDim.y) * blockDim.x + threadIdx.x;
    // Protect memory access with boundary check
    if (i < n) {
        binary[i] = (x[i] >= 0) ? 1.0f : -1.0f;
    }
}