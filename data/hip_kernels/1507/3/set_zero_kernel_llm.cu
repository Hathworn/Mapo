#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_zero_kernel(float *src, int size)
{
    // Calculate the global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride to allow more threads to participate
    int stride = gridDim.x * blockDim.x;

    // Use a loop with stride to ensure all elements are set to zero
    for (; i < size; i += stride)
    {
        src[i] = 0;
    }
}