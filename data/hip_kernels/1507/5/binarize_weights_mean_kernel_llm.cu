#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_mean_kernel(float *weights, int n, int size, float *binary, float *mean_arr_gpu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n * size) { // Calculate boundary condition using a single if
        int f = i / size;
        float mean = mean_arr_gpu[f];
        binary[i] = (weights[i] > 0) ? mean : -mean; // Perform computation directly without additional checks
    }
}