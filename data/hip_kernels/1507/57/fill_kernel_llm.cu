#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x; // Simplified 1D block calculation
    int stride = blockDim.x * gridDim.x; // Calculate stride for grid loops
    for(int i = tid; i < N; i += stride) // Loop to cover all elements
    {
        X[i * INCX] = ALPHA; // Fill array with ALPHA
    }
}