#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mask_kernel_new_api(int n, float *x, float mask_num, float *mask, float val)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we do not go out of bounds
    if (i < n) {
        // Apply mask condition
        if (mask[i] == mask_num) {
            x[i] = val;
        }
    }
}