#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified block index calculation
    if (s >= size) return;

    float mean = 0.0f;
    // Use shared memory for mean calculation
    extern __shared__ float shared[];
    float* subMean = shared + threadIdx.x;

    *subMean = 0.0f;
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        *subMean += fabs(input[i*size + s]);
    }
    __syncthreads();

    // Reduction to compute the mean
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shared[threadIdx.x] += shared[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        mean = shared[0] / n;
    }
    __syncthreads();

    // Update binary array
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}