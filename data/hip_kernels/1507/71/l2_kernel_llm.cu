#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified 1D block index calculation
    if(i < n){
        float diff = truth[i] - pred[i];
        error[i] = diff * diff; // Store squared difference
        delta[i] = diff; // Store difference as delta
    }
}