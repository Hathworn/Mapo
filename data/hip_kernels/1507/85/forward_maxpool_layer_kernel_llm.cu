#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int c = in_c;

    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    int out_index = j + w * (i + h * (k + c * b));
    float max = -INFINITY;
    int max_i = -1;

    // Use registers for loop bounds for efficiency
    #pragma unroll
    for (int l = 0; l < size; ++l) {
        // Precompute h offset for current l iteration
        int cur_h = h_offset + i * stride + l;
        if (cur_h < 0 || cur_h >= in_h) continue;  // Skip invalid rows

        #pragma unroll
        for (int m = 0; m < size; ++m) {
            int cur_w = w_offset + j * stride + m;
            if (cur_w < 0 || cur_w >= in_w) continue;  // Skip invalid columns

            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            float val = input[index];
            if (val > max) {
                max = val;
                max_i = index;
            }
        }
    }
    output[out_index] = max;
    indexes[out_index] = max_i;
}