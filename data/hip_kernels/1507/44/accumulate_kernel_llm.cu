#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;
    
    // Use shared memory for faster accumulation
    extern __shared__ float shared_sum[];
    shared_sum[threadIdx.x] = 0;

    for (int k = 0; k < n; ++k)
    {
        shared_sum[threadIdx.x] += x[k * groups + i];
    }
    
    // Use thread synchronization
    __syncthreads();

    // Write back the result to global memory
    sum[i] = shared_sum[threadIdx.x];
}