#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void repack_input_kernel(float *input, float *re_packed_input, int w, int h, int c)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= w * h * c) return; // Early exit for out-of-bounds threads

    const int items_per_channel = w * h;

    int c_pack = index % 32; // Compute the pack index
    int chan_index = index / 32;
    int chan = (chan_index * 32) % c; // Calculate original channel
    int i = (chan_index * 32) / c; // Linear index within the channel

    if (i < items_per_channel) {
        float src = input[(chan + c_pack) * items_per_channel + i];
        re_packed_input[chan * items_per_channel + i * 32 + c_pack] = src;
    }
}