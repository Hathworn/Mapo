#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_uint32_kernel(uint32_t *src, uint32_t *dst, int src_h, int src_w, int src_align, int dst_align)
{
    // Calculate the column and row for current thread
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure threads are within bounds
    if (j < src_w && i < src_h) {
        // Perform transpose using aligned indices
        dst[j * (dst_align / sizeof(uint32_t)) + i] = src[i * (src_align / sizeof(uint32_t)) + j];
    }
}