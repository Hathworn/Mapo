#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    // Calculate global thread index
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread processes multiple items to increase parallelism
    while (i < N) {
        int out_index = i;
        int out_w = i % (w * stride);
        i = i / (w * stride);
        int out_h = i % (h * stride);
        i = i / (h * stride);
        int out_c = i % c;
        i = i / c;
        int b = i % batch;

        int in_w = out_w / stride;
        int in_h = out_h / stride;
        int in_c = out_c;

        int in_index = b * w * h * c + in_c * w * h + in_h * w + in_w;

        // Use conditional operator for branch optimization
        forward ? out[out_index] += scale * x[in_index]
                : atomicAdd(x + in_index, scale * out[out_index]);

        // Move to next item processed by this thread
        i += blockDim.x * gridDim.x;
    }
}