#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    // Calculate the global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we only process valid ID within bounds
    if(id < size) {
        // Use ternary operation for conditional assignment
        input[id] = (rand[id] < prob) ? 0.0f : input[id] * scale;
    }
}