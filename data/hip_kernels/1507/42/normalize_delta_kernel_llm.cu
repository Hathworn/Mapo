#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (index >= N) return;
    int f = (index / spatial) % filters;

    float inv_sqrt_var = rsqrtf(variance[f] + 1e-6f); // Precompute inverse sqrt
    float factor = 2.f / (spatial * batch);
    
    // Combine multiplications and store intermediary results
    float delta_factor = delta[index] * inv_sqrt_var;
    float variance_contrib = variance_delta[f] * factor * (x[index] - mean[f]);
    float mean_contrib = mean_delta[f] * factor;

    delta[index] = delta_factor + variance_contrib + mean_contrib;
}