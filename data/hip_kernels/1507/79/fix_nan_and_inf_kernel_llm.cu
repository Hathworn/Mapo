#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fix_nan_and_inf_kernel(float *input, size_t size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        float val = input[index];
        // Direct assignment with ternary operator for efficiency
        input[index] = (isnan(val) || isinf(val)) ? (float)index : val;
    }
}