#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n * c) return;

    int k = id % c;
    int b = id / c;

    float sum = 0.0f;
    int out_index = k + c * b;
    for(int i = 0; i < w * h; ++i){
        int in_index = i + w * h * (k + b * c);
        sum += input[in_index]; // Sum up all input values
    }
    output[out_index] = sum / (w * h); // Average the sum
}