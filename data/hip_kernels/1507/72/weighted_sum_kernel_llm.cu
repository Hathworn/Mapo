#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate unique thread index over a 3D block and 3D grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float s_val = s[i];
        float a_val = a[i];
        float b_val = b ? b[i] : 0;
        
        // Optimize by avoiding repeated memory accesses
        c[i] = s_val * a_val + (1 - s_val) * b_val;
    }
}