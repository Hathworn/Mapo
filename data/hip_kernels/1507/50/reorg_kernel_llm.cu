#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Calculate unique global index for each thread
    int index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if(index >= N) return;

    // Compute indices based on flattened index
    int in_index = index;
    int in_w = index % w;
    index /= w;
    int in_h = index % h;
    index /= h;
    int in_c = index % c;
    int b = index / c;

    // Precompute stride-based output dimensions
    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;

    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    // Conditional assignment based on 'forward' flag
    if(forward) {
        out[out_index] = x[in_index];
    } else {
        out[in_index] = x[out_index];
    }
}