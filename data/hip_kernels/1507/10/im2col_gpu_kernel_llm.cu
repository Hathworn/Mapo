#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void im2col_gpu_kernel(const int n, const float* data_im, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_col) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;

    // Use an increment step equivalent to the total number of threads
    for (; index < n; index += total_threads) {
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;

        // Compute column pointer directly
        float* data_col_ptr = data_col + (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im + (channel_in * height + h_in) * width + w_in;

        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                // Use conditional assignment
                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ? 
                                data_im_ptr[i * width + j] : 0;

                data_col_ptr += height_col * width_col; // Move to next depth in column
            }
        }
    }
}