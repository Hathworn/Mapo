#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_input_shortcut_kernel(float *in, int size, float *add, float *out)
{
    // Calculate unique global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize boundary check by processing multiple elements per thread
    int stride = gridDim.x * blockDim.x;

    // Use a loop to process multiple elements, increasing occupancy
    for (; id < size; id += stride) {
        out[id] = in[id] + add[id];
    }
}