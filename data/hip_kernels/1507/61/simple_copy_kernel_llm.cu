#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_copy_kernel(int size, float *src, float *dst)
{
    // Calculate the global thread index efficiently
    int index = blockIdx.x*blockDim.x + threadIdx.x;

    // Check index and copy element using optimal condition
    if (index < size) {
        // Efficient memory access
        dst[index] = src[index];
    }
}