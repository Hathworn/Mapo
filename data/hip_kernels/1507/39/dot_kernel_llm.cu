#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0, norm1 = 0, norm2 = 0;
    // Replace nested loops with a single loop for each batch
    for (int b = 0; b < batch; ++b) {
        int offset = b * size * n;
        for (int i = 0; i < size; ++i) {
            int i1 = offset + f1 * size + i;
            int i2 = offset + f2 * size + i;
            float v1 = output[i1];
            float v2 = output[i2];
            sum += v1 * v2;
            norm1 += v1 * v1;
            norm2 += v2 * v2;
        }
    }
    norm1 = sqrtf(norm1);
    norm2 = sqrtf(norm2);
    float norm = norm1 * norm2;
    sum = sum / norm;
    
    // Combine the updates into a single loop with precomputed indices and values
    for (int b = 0; b < batch; ++b) {
        int offset = b * size * n;
        for (int i = 0; i < size; ++i) {
            int i1 = offset + f1 * size + i;
            int i2 = offset + f2 * size + i;
            float deltaVal = -scale * sum / norm;
            delta[i1] += deltaVal * output[i2];
            delta[i2] += deltaVal * output[i1];
        }
    }
}