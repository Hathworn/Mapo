#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Use shared memory to store biases to reduce global memory access
    __shared__ float shared_biases[1024]; // Assuming max 1024 filters

    if (threadIdx.x == 0) {
        shared_biases[filter] = biases[filter];
    }

    // Synchronize to ensure biases are loaded
    __syncthreads();

    // Use shared bias for multiplication
    if(offset < size) output[(batch * n + filter) * size + offset] *= shared_biases[filter];
}