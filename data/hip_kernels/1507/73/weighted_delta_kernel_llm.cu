#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify the index calculation

    if (i < n) {
        float dc_val = dc[i]; // Load dc[i] once to reduce global memory reads
        float s_val = s[i];   // Load s[i] once to reduce global memory reads

        if (da) da[i] += dc_val * s_val;  // Update da if da is not null

        db[i] += dc_val * (1.0f - s_val); // Use cached s[i] value
        ds[i] += dc_val * (a[i] - b[i]);  // Use cached dc[i] value and operation reordering for clarity
    }
}