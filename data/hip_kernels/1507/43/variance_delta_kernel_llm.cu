#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int filter_id = threadIdx.x + blockIdx.x * blockDim.x; // Direct thread index calculation
    if (filter_id >= filters) return;

    float variance_value = variance[filter_id] + 0.000001f;
    float inv_variance = -0.5f * powf(variance_value, -1.5f); // Pre-compute invariant part

    for (int j = 0; j < batch; ++j)
    {
        for (int k = 0; k < spatial; ++k)
        {
            int index = j * filters * spatial + filter_id * spatial + k;
            variance_delta[filter_id] += delta[index] * (x[index] - mean[filter_id]);
        }
    }
    variance_delta[filter_id] *= inv_variance; // Efficient multiplication with pre-computed value
}