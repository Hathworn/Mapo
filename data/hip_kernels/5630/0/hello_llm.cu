#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use blockIdx.x and threadIdx.x to identify the thread. This optimizes the thread's printing operation.
    printf("Hello world! I'm thread %d in block %d\n", threadIdx.x, blockIdx.x);
}