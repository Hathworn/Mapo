#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void sieveOfEratosthenes(char *primes, uint64_t max) {
    // Optimize for loop conditions and access patterns
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index == 0 && threadIdx.y == 0) {
        primes[0] = 1; // 0 is not prime
        primes[1] = 1; // 1 is not prime
        // Sieve multiples of two
        for (uint64_t j = 4; j < max; j += 2) {
            primes[j] = 1;
        }
        return;
    }

    const uint64_t maxRoot = sqrt((double)max);
    if (index <= maxRoot && index > 1 && primes[index] == 0) {
        // Mark off composite numbers
        for (uint64_t j = index * index; j < max; j += index) {
            primes[j] = 1;
        }
    }
}