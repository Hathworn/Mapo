#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sieveOfEratosthenesCUDA(char *primes, uint64_t max) {
    // Optimize by reducing thread divergence
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    const uint64_t maxRoot = sqrt((double)max);

    if (index == 0) {
        primes[0] = 1; // 0 is not prime
        primes[1] = 1; // 1 is not prime
        // mark multiples of 2
        for (int j = 2 * 2; j < max; j += 2) {
            primes[j] = 1;
        }
        return; // exit for index 0 after initialization
    }

    if (index < maxRoot && primes[index] == 0 && index > 1) {
        // mark off composite numbers
        for (int j = index * index; j < max; j += index) {
            primes[j] = 1;
        }
    }
}