#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void segmentSieve(char *primes, uint64_t max) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for efficient data access
    extern __shared__ char shared_primes[];

    if (index > 0) {
        const uint64_t maxRoot = sqrt((double)max);
        int low = maxRoot * index;
        int high = low + maxRoot;
        if (high > max) high = max;
        
        // Initialize shared memory
        for (int i = threadIdx.x; i < maxRoot; i += blockDim.x) {
            shared_primes[i] = primes[i];
        }
        __syncthreads();

        for (int i = 2; i < maxRoot; i++) {
            if (shared_primes[i] == 0) { // Access shared memory
                int loLim = (low / i) * i;
                if (loLim < low)
                    loLim += i;
                for (int j = loLim; j < high; j += i)
                    primes[j] = 1;
            }
        }
    }
}