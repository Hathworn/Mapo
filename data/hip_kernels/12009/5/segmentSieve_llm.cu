#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void segmentSieve(char *primes, uint64_t max) {
    // Simplified index calculation
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index > 0) {
        const uint64_t maxRoot = sqrt((double)max);
        // Calculate low and high efficiently
        uint64_t low = maxRoot * index;
        uint64_t high = min(low + maxRoot, max);

        // Use shared memory for maxRoot primes
        __shared__ char sharedPrimes[1024]; // Adjust size as needed
        if (threadIdx.x < maxRoot) {
            sharedPrimes[threadIdx.x] = primes[threadIdx.x];
        }
        __syncthreads();

        for (uint64_t i = 2; i < maxRoot; i++) {
            if(sharedPrimes[i] == 0) { // Use shared memory
                // Calculate the loLim more accurately
                uint64_t loLim = max(low / i * i, (uint64_t)i * i);
                if (loLim < low)
                    loLim += i;

                // Unroll the loop for efficiency
                for (uint64_t j = loLim; j < high; j += i) {
                    primes[j] = 1;
                }
            }
        }
    }
}