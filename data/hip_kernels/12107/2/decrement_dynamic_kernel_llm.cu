#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void decrement_dynamic_kernel(int* pInts, size_t numInts)
{
    // Calculate global thread index
    size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Optimize boundary condition check
    if (idx < numInts) {
        pInts[idx] -= 1;
    }
}