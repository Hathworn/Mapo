#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_dynamic_kernel(const int* pIn, int* pOut, size_t numInts)
{
    extern __shared__ int ps[]; // Declare shared memory

    // Loading shared memory with boundary checks
    const unsigned int tid = threadIdx.x;
    unsigned int idx = tid + blockIdx.x * blockDim.x;
    if (idx < numInts) 
    {
        ps[tid] = pIn[idx];
    }
    else 
    {
        ps[tid] = 0; // Handle out-of-bound threads
    }

    __syncthreads(); // Synchronize to ensure shared memory is fully loaded

    // Reduce within shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) 
    {
        if (tid < s) 
        {
            ps[tid] += ps[tid + s];
        }
        __syncthreads(); // Synchronize to ensure all threads have updated values
    }

    // Write result for this block to global memory
    if (tid == 0) 
    {
        atomicAdd(pOut, ps[0]); // Use atomic operation to avoid race conditions
    }
}