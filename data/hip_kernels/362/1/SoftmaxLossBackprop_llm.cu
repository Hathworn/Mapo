#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* This code is released into the public domain.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
* EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
* MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
* IN NO EVENT SHALL THE AUTHORS BE LIABLE FOR ANY CLAIM, DAMAGES OR
* OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE,
* ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
* OTHER DEALINGS IN THE SOFTWARE.
*/

///////////////////////////////////////////////////////////////////////////////////////////
// Definitions and helper utilities

// Block width for CUDA kernels
#define BW 128

#ifdef USE_GFLAGS

#ifndef _WIN32
#define gflags google
#endif
#else
// Constant versions of gflags
#define DEFINE_int32(flag, default_value, description) const int FLAGS_##flag = (default_value)
#define DEFINE_uint64(flag, default_value, description) const unsigned long long FLAGS_##flag = (default_value)
#define DEFINE_bool(flag, default_value, description) const bool FLAGS_##flag = (default_value)
#define DEFINE_double(flag, default_value, description) const double FLAGS_##flag = (default_value)
#define DEFINE_string(flag, default_value, description) const std::string FLAGS_##flag ((default_value))
#endif

/**
* Computes ceil(x / y) for integral nonnegative values.
*/
__global__ void SoftmaxLossBackprop(const float *label, int num_labels, int batch_size, float *diff)
{
    // Calculate index based on grid and block index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Make sure idx is valid
    if (idx < batch_size) {
        // Fetch and cast the label value
        const int label_value = static_cast<int>(label[idx]);
        
        // Update diff using atomic operation to prevent race condition
        atomicSub(&diff[idx * num_labels + label_value], 1.0f); 
    }
}