#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Block width for CUDA kernels
#define BW 128

#ifdef USE_GFLAGS

#ifndef _WIN32
#define gflags google
#endif
#else
// Constant versions of gflags
#define DEFINE_int32(flag, default_value, description) const int FLAGS_##flag = (default_value)
#define DEFINE_uint64(flag, default_value, description) const unsigned long long FLAGS_##flag = (default_value)
#define DEFINE_bool(flag, default_value, description) const bool FLAGS_##flag = (default_value)
#define DEFINE_double(flag, default_value, description) const double FLAGS_##flag = (default_value)
#define DEFINE_string(flag, default_value, description) const std::string FLAGS_##flag ((default_value))
#endif

/**
* Computes ceil(x / y) for integral nonnegative values.
*/
__global__ void FillOnes(float *vec, int size)
{
    // Optimize by using shared memory
    __shared__ float sharedVec[BW];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        sharedVec[threadIdx.x] = 1.0f;
        vec[idx] = sharedVec[threadIdx.x];
    }
}