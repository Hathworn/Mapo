#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv2genericrev(float *input, float *kernel, float *output, int input_n, int input_h, int input_w, int kernel_n, int kernel_h, int kernel_w, float alpha, int stride_h, int stride_w)
{
    // output dimensions
    int output_h = input_h - (kernel_h - 1) * stride_h;
    int output_w = input_w - (kernel_w - 1) * stride_w;

    // Calculate index positions
    int kk = blockIdx.x;
    int ii = blockIdx.y;
    int batch = threadIdx.z;
    int kid = threadIdx.x;
    int nkids = blockDim.x;
    int tid = kid + batch * nkids;
    int nthreads = blockDim.x * blockDim.z;
    
    output += (kk * input_n + ii) * output_h * output_w;
    
    // Shared memory for output
    extern __shared__ float shared_output[];

    // Position in shared memory
    float *output_s = &shared_output[tid * output_w * output_h];

    // Set y once, re-use
    int yy = threadIdx.y;
    float *out = output_s + yy * output_w;

    // Convolution loop
    float *input_p = input + (ii + batch * input_n) * input_h * input_w + yy * stride_h * input_w;
    float *kernel_p = kernel + (kk + batch * kernel_n) * kernel_w * kernel_h;

    for (int xx = 0; xx < output_w; xx++) {
        float sum = 0.0f;
        for (int ky = 0; ky < kernel_h; ky++) {
            for (int kx = kid; kx < kernel_w; kx += nkids) {
                sum += input_p[kx] * kernel_p[kx];
            }
            input_p += input_w;
            kernel_p += kernel_w;
        }
        out[xx] = sum;
    }
    __syncthreads();
    
    // Reduction and output write-back, executed by first thread in each group
    if (yy == 0) {
        for (int k = 1; k < nthreads; k++) {
            for (int i = tid; i < output_w * output_h; i += nthreads) {
                shared_output[i] += shared_output[k * output_w * output_h + i];
            }
        }
        __syncthreads();
        
        // Write back results
        for (int i = tid; i < output_w * output_h; i += nthreads) {
            output[i] += alpha * shared_output[i];
        }
    }
}