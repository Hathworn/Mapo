#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "max_norm_matrix_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *mat1 = NULL;
hipMalloc(&mat1, XSIZE*YSIZE);
int row = 1;
int col = 1;
int *norm = NULL;
hipMalloc(&norm, XSIZE*YSIZE);
float *final_norm = NULL;
hipMalloc(&final_norm, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
max_norm_matrix<<<gridBlock, threadBlock>>>(mat1,row,col,norm,final_norm);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
max_norm_matrix<<<gridBlock, threadBlock>>>(mat1,row,col,norm,final_norm);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
max_norm_matrix<<<gridBlock, threadBlock>>>(mat1,row,col,norm,final_norm);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}