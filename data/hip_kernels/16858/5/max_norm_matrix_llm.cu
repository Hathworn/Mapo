#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void max_norm_matrix(float* mat1, int row, int col, int* norm, float* final_norm) {
    // Use shared memory for faster atomic operations
    extern __shared__ int shared_norm[];
    int tid = threadIdx.x;
    shared_norm[tid] = 0; // Initialize shared memory
    __syncthreads();

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = row * col;
    
    if (id < size) {
        atomicMax(&shared_norm[tid], __float_as_int(abs(mat1[id])));
    }
    __syncthreads();

    // Use one thread to update the global norm based on shared memory
    if (tid == 0) {
        for (int i = 0; i < blockDim.x; ++i) {
            atomicMax(norm, shared_norm[i]);
        }
    }
    __syncthreads();

    // Use the first thread to compute the final norm
    if (id == 0) {
        *final_norm = __int_as_float(*norm);
    }
}