#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_add_matrix(float* mat1, float* mat2, float* mat3, int row, int col, int sign) {
    // Calculate global index using block and thread indices
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = row * col;
    
    // Perform vectorized addition if within bounds
    if (id < size) {
        mat3[id] = mat1[id] + sign * mat2[id];
    }
}