#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_mul_matrix(float *A, float *B, float *C, int col_A, int col_B, int row_C, int col_C) {
    // Shared memory for storing tile data
    __shared__ float tile_A[32][32]; // Use appropriate size based on block dimensions
    __shared__ float tile_B[32][32];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    for (int tile_idx = 0; tile_idx < col_A; tile_idx += blockDim.x) {
        // Load data into shared memory
        if (row < row_C && (tile_idx + threadIdx.x) < col_A)
            tile_A[threadIdx.y][threadIdx.x] = A[row * col_A + (tile_idx + threadIdx.x)];
        else
            tile_A[threadIdx.y][threadIdx.x] = 0;

        if (col < col_C && (tile_idx + threadIdx.y) < col_B)
            tile_B[threadIdx.y][threadIdx.x] = B[(tile_idx + threadIdx.y) * col_B + col];
        else
            tile_B[threadIdx.y][threadIdx.x] = 0;

        __syncthreads();

        // Compute partial product within the tile
        for (int i = 0; i < blockDim.x; ++i) {
            sum += tile_A[threadIdx.y][i] * tile_B[i][threadIdx.x];
        }
        __syncthreads();
    }

    // Write the result to global memory
    if (row < row_C && col < col_C) {
        C[row * col_B + col] = sum;
    }
}