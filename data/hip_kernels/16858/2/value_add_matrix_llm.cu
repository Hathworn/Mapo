#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void value_add_matrix(float* __restrict__ mat1, float* __restrict__ mat2, int row, int col, float v) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int size = row * col;
    
    // Loop to handle sizes larger than the current block/grid configuration
    for (int i = id; i < size; i += stride) {
        mat2[i] = mat1[i] + v;  // Efficient memory access pattern
    }
}