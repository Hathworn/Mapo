#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void seq_max_norm(float* mat1, int row, int col, float* norm) {
    // Utilize shared memory for partial results
    __shared__ float shared_norm;
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;

    // Initialize shared memory to zero
    if (threadIdx.x == 0) shared_norm = 0.0f;
    __syncthreads();

    // Calculate maximum in parallel
    float local_max = 0.0f;
    for (int idx = tid; idx < row * col; idx += num_threads) {
        local_max = max(abs(mat1[idx]), local_max);
    }

    // Reduce local maxima to shared memory
    atomicMax(&shared_norm, local_max);
    __syncthreads();

    // Update global norm from shared memory
    if (threadIdx.x == 0) {
        atomicMax(norm, shared_norm);
    }
}