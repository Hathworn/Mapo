#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_mat(float* mat, int row, int col) {
    // Calculate the global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use only one thread to print the matrix
    if (id == 0) {
        for (int i = 0; i < row; ++i) {
            for (int j = 0; j < col; ++j) {
                // More efficient printing format
                printf("%.3f\t", mat[i * col + j]);
            }
            printf("\n");
        }
        printf("\n");
    }
}