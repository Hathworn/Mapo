#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void value_mul_matrix(float* mat1, float* mat2, int row, int col, float v) {
    // Calculate unique thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = row * col;
    // Ensure work is done only for valid thread indices
    if (id < size) {
        mat2[id] = mat1[id] * v;
    }
}