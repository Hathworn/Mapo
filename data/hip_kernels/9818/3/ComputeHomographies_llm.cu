#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//================= Device matching functions =====================//

template <int size>
__device__ void InvertMatrix(float elem[size][size], float res[size][size])
{
    int indx[size];
    float b[size];
    float vv[size];
    for (int i = 0; i < size; i++)
        indx[i] = 0;
    int imax = 0;
    float d = 1.0;
    for (int i = 0; i < size; i++) { // find biggest element for each row
        float big = 0.0;
        for (int j = 0; j < size; j++) {
            float temp = fabs(elem[i][j]);
            if (temp > big)
                big = temp;
        }
        if (big > 0.0)
            vv[i] = 1.0 / big;
        else
            vv[i] = 1e16;
    }
    for (int j = 0; j < size; j++) {
        for (int i = 0; i < j; i++) { // i<j
            float sum = elem[i][j]; // i<j (lower left)
            for (int k = 0; k < i; k++) // k<i<j
                sum -= elem[i][k] * elem[k][j]; // i>k (upper right), k<j (lower left)
            elem[i][j] = sum; // i<j (lower left)
        }
        float big = 0.0;
        for (int i = j; i < size; i++) { // i>=j
            float sum = elem[i][j]; // i>=j (upper right)
            for (int k = 0; k < j; k++) // k<j<=i
                sum -= elem[i][k] * elem[k][j]; // i>k (upper right), k<j (lower left)
            elem[i][j] = sum; // i>=j (upper right)
            float dum = vv[i] * fabs(sum);
            if (dum >= big) {
                big = dum;
                imax = i;
            }
        }
        if (j != imax) { // imax>j
            for (int k = 0; k < size; k++) {
                float dum = elem[imax][k]; // upper right and lower left
                elem[imax][k] = elem[j][k];
                elem[j][k] = dum;
            }
            d = -d;
            vv[imax] = vv[j];
        }
        indx[j] = imax;
        if (elem[j][j] == 0.0) // j==j (upper right)
            elem[j][j] = 1e-16;
        if (j != (size - 1)) {
            float dum = 1.0 / elem[j][j];
            for (int i = j + 1; i < size; i++) // i>j
                elem[i][j] *= dum; // i>j (upper right)
        }
    }
    for (int j = 0; j < size; j++) {
        for (int k = 0; k < size; k++)
            b[k] = 0.0;
        b[j] = 1.0;
        int ii = -1;
        for (int i = 0; i < size; i++) {
            int ip = indx[i];
            float sum = b[ip];
            b[ip] = b[i];
            if (ii != -1)
                for (int j = ii; j < i; j++)
                    sum -= elem[i][j] * b[j]; // i>j (upper right)
            else if (sum != 0.0)
                ii = i;
            b[i] = sum;
        }
        for (int i = size - 1; i >= 0; i--) {
            float sum = b[i];
            for (int j = i + 1; j < size; j++)
                sum -= elem[i][j] * b[j]; // i<j (lower left)
            b[i] = sum / elem[i][i]; // i==i (upper right)
        }
        for (int i = 0; i < size; i++)
            res[i][j] = b[i];
    }
}

__global__ void ComputeHomographies(float* coord, int* randPts, float* homo, int numPts)
{
    float a[8][8], ia[8][8];
    float b[8];
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    const int numLoops = blockDim.x * gridDim.x;

    // Avoid branch divergence by using the same loop across threads
    for (int i = 0; i < 4; i++) {
        int pt = randPts[i * numLoops + idx];
        float x1 = coord[pt];
        float y1 = coord[pt + numPts];
        float x2 = coord[pt + 2 * numPts];
        float y2 = coord[pt + 3 * numPts];

        // Precompute repeated values to reduce redundant calculations
        float neg_x2_x1 = -x2 * x1;
        float neg_x2_y1 = -x2 * y1;
        float neg_y2_x1 = -y2 * x1;
        float neg_y2_y1 = -y2 * y1;

        // Populate matrices
        a[2 * i][0] = x1; a[2 * i][1] = y1; a[2 * i][2] = 1.0f;
        a[2 * i][3] = a[2 * i][4] = a[2 * i][5] = 0.0f;
        a[2 * i][6] = neg_x2_x1; a[2 * i][7] = neg_x2_y1;
        
        a[2 * i + 1][0] = a[2 * i + 1][1] = a[2 * i + 1][2] = 0.0f;
        a[2 * i + 1][3] = x1; a[2 * i + 1][4] = y1; a[2 * i + 1][5] = 1.0f;
        a[2 * i + 1][6] = neg_y2_x1; a[2 * i + 1][7] = neg_y2_y1;

        // Vector b with destination coordinates
        b[2 * i] = x2;
        b[2 * i + 1] = y2;
    }
    
    // Perform matrix inversion
    InvertMatrix<8>(a, ia);
    __syncthreads(); // Ensure all threads reach this point before proceeding

    // Compute homography matrix
    for (int j = 0; j < 8; j++) {
        float sum = 0.0f;
        for (int i = 0; i < 8; i++)
            sum += ia[j][i] * b[i];
        homo[j * numLoops + idx] = sum;
    }
    __syncthreads(); // Ensure all threads have finished computation
}