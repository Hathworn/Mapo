#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BW (SCALEUP_W/2 + 2)
#define BH (SCALEUP_H/2 + 2)

__global__ void ScaleUp(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
    __shared__ float buffer[BW*BH];

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int blockX = blockIdx.x * (SCALEUP_W / 2);
    const int blockY = blockIdx.y * (SCALEUP_H / 2);

    // Optimize shared memory loading with fewer conditionals
    if (tx < BW && ty < BH) {
        int x = min(max(blockX + tx - 1, 0), width - 1);
        int y = min(max(blockY + ty - 1, 0), height - 1);
        buffer[ty * BW + tx] = d_Data[y * pitch + x];
    }
    
    __syncthreads();

    int x = blockIdx.x * SCALEUP_W + tx;
    int y = blockIdx.y * SCALEUP_H + ty;

    // Use pre-computed indices and weights for simplicity
    if (x < 2 * width && y < 2 * height) {
        int bx = (tx + 1) >> 1;
        int by = (ty + 1) >> 1;
        int bp = by * BW + bx;
        float wx = 0.25f + (tx & 1) * 0.50f;
        float wy = 0.25f + (ty & 1) * 0.50f;

        d_Result[y * newpitch + x] = wy * (wx * buffer[bp] + (1.0f - wx) * buffer[bp + 1]) +
                                     (1.0f - wy) * (wx * buffer[bp + BW] + (1.0f - wx) * buffer[bp + BW + 1]);
    }
}