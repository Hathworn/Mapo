#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuComputeDistanceGlobal(float* A, int wA, float* B, int wB, int dim, float* AB) {

    // Shared memory for sub-matrix of A and B
    __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Block index for A and B
    int aBegin = BLOCK_DIM * blockIdx.y;
    int bBegin = BLOCK_DIM * blockIdx.x;
    int aEnd = aBegin + (dim - 1) * wA;
    int aStep = BLOCK_DIM * wA;
    int bStep = BLOCK_DIM * wB;

    // For storage and computation
    float ssd = 0.0;
    float tmp;

    // Load and compute loop
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        // Load elements from global memory to shared memory
        shared_A[ty][tx] = (a / wA + ty < dim && a + wA * ty + tx < wA * dim) ? A[a + wA * ty + tx] : 0.0f;
        shared_B[ty][tx] = (b + wB * ty + tx < wB * dim) ? B[b + wB * ty + tx] : 0.0f;

        // Synchronize threads to ensure all data is loaded
        __syncthreads();

        // Compute the sum of squared differences
        #pragma unroll
        for (int k = 0; k < BLOCK_DIM; ++k) {
            tmp = shared_A[k][ty] - shared_B[k][tx];
            ssd += tmp * tmp;
        }

        // Synchronize threads before next iteration
        __syncthreads();
    }

    // Write computed result back to global memory
    if ((aBegin + ty < wA) && (bBegin + tx < wB)) {
        AB[(aBegin + ty) * wB + bBegin + tx] = ssd;
    }
}