#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuInsertionSort(float *dist, long *ind, int width, int height, int k) {
    // Optimize variable usage
    int i, j;
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (xIndex < width) {
        // Use shared memory to reduce global memory access
        extern __shared__ float s_dist[];
        extern __shared__ long s_ind[];

        // Initialize and copy global to shared memory
        float max_dist = dist[xIndex];
        s_dist[0] = max_dist;
        s_ind[0] = 1;

        // Part 1: Sort first k elements
        for (int l = 1; l < k; l++) {
            float curr_dist = dist[l * width + xIndex];
            long curr_row = l * width + xIndex;

            if (curr_dist < max_dist) {
                i = l - 1;
                for (int a = 0; a < l - 1; a++) {
                    if (s_dist[a] > curr_dist) {
                        i = a;
                        break;
                    }
                }
                // Optimize shift operation using shared memory
                for (j = l; j > i; j--) {
                    s_dist[j] = s_dist[j - 1];
                    s_ind[j] = s_ind[j - 1];
                }
                s_dist[i] = curr_dist;
                s_ind[i] = l + 1;
            } else {
                s_ind[l] = l + 1;
            }
            max_dist = s_dist[l];
        }

        // Copy sorted results back to global memory for first k items
        for (j = 0; j < k; j++) {
            dist[j * width + xIndex] = s_dist[j];
            ind[j * width + xIndex] = s_ind[j];
        }

        // Part 2: Insert elements in the k-th first lines
        float max_row = s_dist[k - 1];
        for (int l = k; l < height; l++) {
            float curr_dist = dist[l * width + xIndex];

            if (curr_dist < max_row) {
                i = k - 1;
                for (int a = 0; a < k - 1; a++) {
                    if (s_dist[a] > curr_dist) {
                        i = a;
                        break;
                    }
                }
                // Optimize shift operation using shared memory
                for (j = k - 1; j > i; j--) {
                    s_dist[j] = s_dist[j - 1];
                    s_ind[j] = s_ind[j - 1];
                }
                s_dist[i] = curr_dist;
                s_ind[i] = l + 1;

                max_row = s_dist[k - 1];
            }
        }

        // Copy sorted results back to global memory for entire height
        for (j = 0; j < k; j++) {
            dist[j * width + xIndex] = s_dist[j];
            ind[j * width + xIndex] = s_ind[j];
        }
    }
}