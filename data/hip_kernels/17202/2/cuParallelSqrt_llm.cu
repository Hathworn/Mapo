#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuParallelSqrt(float *dist, int width, int k) {
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    // Use a single combined index for better memory access patterns
    unsigned int index = yIndex * width + xIndex;
    if (xIndex < width && yIndex < k) {
        // Save calculation of index in a variable, avoid recalculating.
        float value = dist[index];
        dist[index] = sqrt(value);
    }
}