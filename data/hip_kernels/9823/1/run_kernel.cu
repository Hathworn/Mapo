#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef ENABLE_CUDA
#pragma GCC diagnostic push
#pragma GCC diagnostic pop
#endif


#define SIZE 256



__global__ void run_kernel(hiprandStateMRG32k3a *state, unsigned int *result) {
int id = threadIdx.x + blockIdx.x * SIZE;
hiprandStateMRG32k3a localState = state[id];
unsigned int x = hiprand(&localState);
while (x == 0) {
x = hiprand(&localState);
}
state[id] = localState;
result[id] = x;
}