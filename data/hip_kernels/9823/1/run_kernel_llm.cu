#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef ENABLE_CUDA
#pragma GCC diagnostic push
#pragma GCC diagnostic pop
#endif

#define SIZE 256

__global__ void run_kernel(hiprandStateMRG32k3a *state, unsigned int *result) {
    int id = threadIdx.x + blockIdx.x * SIZE;
    
    // Load state only once per thread
    hiprandStateMRG32k3a localState = state[id];
    
    // Generate random number until a non-zero value is found
    unsigned int x;
    do {
        x = hiprand(&localState);
    } while (x == 0);
    
    // Store back the updated state and the result
    state[id] = localState;
    result[id] = x;
}