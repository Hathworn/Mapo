#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef ENABLE_CUDA
#pragma GCC diagnostic push
#pragma GCC diagnostic pop
#endif


#define SIZE 256



__global__ void setup_kernel(hiprandStateMRG32k3a *state, unsigned int seed) {
int id = threadIdx.x + blockIdx.x * SIZE;
hiprand_init(seed, id, 0, &state[id]);
}