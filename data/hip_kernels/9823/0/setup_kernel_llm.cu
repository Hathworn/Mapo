#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef ENABLE_CUDA
#pragma GCC diagnostic push
#pragma GCC diagnostic pop
#endif

#define SIZE 256

__global__ void setup_kernel(hiprandStateMRG32k3a *state, unsigned int seed) {
    // Using blockDim to accommodate any grid size and improve scalability
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &state[id]);
}