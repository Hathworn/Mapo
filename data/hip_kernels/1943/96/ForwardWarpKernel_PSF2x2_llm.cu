#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    
    // Pixel containing bottom left corner
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    
    // Target pixel integer coords
    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];

    // Cache computations to avoid repeated calculations
    float dx1 = 1.0f - dx;
    float dy1 = 1.0f - dy;

    // Fill pixel containing bottom right corner
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        float weight = dx * dy;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Fill pixel containing bottom left corner
    tx -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        float weight = dx1 * dy;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Fill pixel containing upper left corner
    ty -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        float weight = dx1 * dy1;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Fill pixel containing upper right corner
    tx += 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        float weight = dx * dy1;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}