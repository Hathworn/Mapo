#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate the linear index for the thread
    int pos = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Use shared memory cautiously to improve caching (useful in larger blocks)
    __shared__ float sharedValue;
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        sharedValue = value;
    }
    __syncthreads();

    // Boundary check and assignment
    if (pos < h && j < w) {
        image[pos * w + j] = sharedValue; // Strided write for coalesced memory access
    }
}