#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure active thread processes valid points only
    if (ptidx >= npoints)
        return;

    short2 loc = loc_[ptidx];

    // Use FMAD for potential performance gain
    x[ptidx] = __fmul_rn(loc.x, scale);
    y[ptidx] = __fmul_rn(loc.y, scale);
}