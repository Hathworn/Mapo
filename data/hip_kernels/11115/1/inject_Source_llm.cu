#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define C0  0
#define CZ1 1
#define CX1 2
#define CY1 3
#define CZ2 4
#define CX2 5
#define CY2 6
#define CZ3 7
#define CX3 8
#define CY3 9
#define CZ4 10
#define CX4 11
#define CY4 12

__global__ void inject_Source(int id, int ii, float *p, float *_tableS, float *_sourceV, int *_locsS, int _dir, int _jt, int _ntSrc){
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    // Check bounds to prevent memory access errors
    if (ix < _ntSrc) {
        float partial_sum = 0.0f;

        // Unroll loop for performance improvement
        #pragma unroll
        for (int offset = 0; offset < 8; offset++) {
            partial_sum += _tableS[ii + offset] * _sourceV[_ntSrc * ix + id + offset];
        }

        // Atomic operations to avoid race conditions
        atomicAdd(&p[_locsS[ix]], (float)_dir / _jt * partial_sum);
    }
}