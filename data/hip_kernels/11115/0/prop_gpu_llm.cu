#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define C0  0
#define CZ1 1
#define CX1 2
#define CY1 3
#define CZ2 4
#define CX2 5
#define CY2 6
#define CZ3 7
#define CX3 8
#define CY3 9
#define CZ4 10
#define CX4 11
#define CY4 12

__global__ void prop_gpu(float *p0, float *p1, float *vel, float *coeffs, int _nx, int _ny, int _nz, int _n12) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure boundary conditions are met before proceeding
    if (col >= 4 && col < _nx - 4 && row >= 4 && row < _ny - 4) {
        int ii = row * _nx + col;
        int stride_x = 1;
        int stride_y = _nx;
        int stride_z = _n12;

        for (int z = 4; z < _nz - 4; z++) {
            float val = coeffs[C0] * p1[ii];
            // Compute contributions along X, Y, and Z separately
            for (int offset = 1; offset <= 4; offset++) {
                val += coeffs[offset] * (p1[ii - offset * stride_x] + p1[ii + offset * stride_x]);
                val += coeffs[offset + 4] * (p1[ii - offset * stride_y] + p1[ii + offset * stride_y]);
                val += coeffs[offset + 8] * (p1[ii - offset * stride_z] + p1[ii + offset * stride_z]);
            }
            p0[ii] = vel[ii] * val + 2 * p1[ii] - p0[ii];
            ii += stride_z;
        }
    }
}
```
