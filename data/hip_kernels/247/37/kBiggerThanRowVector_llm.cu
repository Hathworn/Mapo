#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBiggerThanRowVector(float* mat, float* vec, float* tgtMat, const int width, const int height) {
    // Calculate the row and column indices directly
    const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we are within matrix bounds
    if (row < height && col < width) {
        unsigned int i = row * width + col;
        tgtMat[i] = mat[i] > vec[col];
    }
}