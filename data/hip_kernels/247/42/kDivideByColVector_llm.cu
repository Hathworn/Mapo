#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivideByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate thread's 2D position within the matrix
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    // Optimize loop to directly handle data assigned to each thread
    for (unsigned int i = col; i < width * height; i += stride) {
        // Divide each element in the column by the corresponding vector element
        tgtMat[i] = __fdividef(mat[i], vec[i / width]);
    }
}