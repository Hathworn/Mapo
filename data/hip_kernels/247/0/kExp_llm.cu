#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kExp(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use unrolled loop for better performance
    unsigned int stride = blockDim.x * gridDim.x;
    for (unsigned int i = idx; i + 3 < numElements; i += 4 * stride) { 
        target[i] = __expf(gData[i]);
        if (i + stride < numElements) target[i + stride] = __expf(gData[i + stride]);
        if (i + 2 * stride < numElements) target[i + 2 * stride] = __expf(gData[i + 2 * stride]);
        if (i + 3 * stride < numElements) target[i + 3 * stride] = __expf(gData[i + 3 * stride]);
    }
}