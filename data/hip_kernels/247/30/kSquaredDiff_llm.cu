#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSquaredDiff(float* a, float* b, float* dest, unsigned int numEls) {
    // Optimize thread index calculation
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    // Unroll the loop to reduce the number of iterations
    for (; i + 6 < numEls; i += 7 * stride) {
        dest[i] = (a[i] - b[i]) * (a[i] - b[i]);
        dest[i + stride] = (a[i + stride] - b[i + stride]) * (a[i + stride] - b[i + stride]);
        dest[i + 2 * stride] = (a[i + 2 * stride] - b[i + 2 * stride]) * (a[i + 2 * stride] - b[i + 2 * stride]);
        dest[i + 3 * stride] = (a[i + 3 * stride] - b[i + 3 * stride]) * (a[i + 3 * stride] - b[i + 3 * stride]);
        dest[i + 4 * stride] = (a[i + 4 * stride] - b[i + 4 * stride]) * (a[i + 4 * stride] - b[i + 4 * stride]);
        dest[i + 5 * stride] = (a[i + 5 * stride] - b[i + 5 * stride]) * (a[i + 5 * stride] - b[i + 5 * stride]);
        dest[i + 6 * stride] = (a[i + 6 * stride] - b[i + 6 * stride]) * (a[i + 6 * stride] - b[i + 6 * stride]);
    }

    // Handle remaining elements
    for (; i < numEls; i += stride) {
        dest[i] = (a[i] - b[i]) * (a[i] - b[i]);
    }
}