#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numCols, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;
    if (eidx < numCols) {
        for (; eidx < numElements; eidx += numCols) {
            sum += a[eidx] * b[eidx];
        }
    }
    shmem[threadIdx.x] = sum; // Store sum in shared memory
    __syncthreads();

    // Unroll loop for reduction
    for (int offset = DP_BLOCKSIZE / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            shmem[threadIdx.x] += shmem[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}