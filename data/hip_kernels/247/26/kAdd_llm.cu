#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd(float* a, float* b, float* dest, unsigned int numEls, float scaleA, float scaleB) {
    // Calculate global thread index for 1D grid
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread processes multiple elements using a strided approach
    for (unsigned int i = idx; i < numEls; i += blockDim.x * gridDim.x) {
        dest[i] = scaleA * a[i] + scaleB * b[i];
    }
}