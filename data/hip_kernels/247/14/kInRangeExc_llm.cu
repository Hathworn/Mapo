#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kInRangeExc(float* gData, float lower, float upper, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Process elements with a stride loop for better balance
    for (unsigned int i = idx; i < numElements; i += stride) {
        target[i] = (gData[i] > lower) && (gData[i] < upper);
    }
}