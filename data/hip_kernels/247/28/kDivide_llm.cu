#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivide(float* a, float* b, float* dest, unsigned int numEls) {
    // Simplified computation of the global thread index
    const unsigned int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    // Retrieve total number of threads used in the kernel
    const unsigned int totalThreads = blockDim.x * gridDim.x;

    // Use stride loop for optimal memory access and work distribution
    for (unsigned int i = globalIdx; i < numEls; i += totalThreads) {
        dest[i] = __fdividef(a[i], b[i]); // Efficient division using intrinsic function
    }
}