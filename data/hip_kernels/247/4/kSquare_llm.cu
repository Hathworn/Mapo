#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSquare(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Utilize shared memory to reduce global memory access
    __shared__ float sharedData[1024]; // Adjust size based on your block size

    if (idx < numElements) {
        sharedData[threadIdx.x] = gData[idx];
    }
    __syncthreads();

    // Process elements using shared memory
    for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x) {
        if (i < numElements) {
            target[i] = sharedData[threadIdx.x] * sharedData[threadIdx.x];
        }
    }
}