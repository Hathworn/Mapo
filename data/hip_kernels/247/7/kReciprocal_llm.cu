#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kReciprocal(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread processes valid indices only.
    if (idx < numElements) {
        target[idx] = 1 / gData[idx];
    }
}