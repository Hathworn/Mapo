#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float computeDeterminant(float e00, float e01, float e02, float e10, float e11, float e12, float e20, float e21, float e22) {
    return e00 * e11 * e22 - e00 * e12 * e21 + e10 * e21 * e02 - e10 * e01 * e22 + e20 * e01 * e12 - e20 * e11 * e02;
}

__global__ void hessianKernelO(float *d_output, float *d_output_theta, float *d_output_phi, const float *d_gxx, const float *d_gxy, const float *d_gxz, const float *d_gyy, const float *d_gyz, const float *d_gzz, float sigma, int imageW, int imageH, int imageD) {
    int n_blocks_per_width = imageW / blockDim.x;
    int z = blockIdx.x / n_blocks_per_width; // Simplified calculation for z
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = (blockIdx.x % n_blocks_per_width) * blockDim.x + threadIdx.x; // Simplified calculation for x
    int i = z * imageW * imageH + y * imageW + x;

    float a0 = -d_gxx[i];
    float b0 = -d_gxy[i];
    float c0 = -d_gxz[i];
    float d0 = -d_gyy[i];
    float e0 = -d_gyz[i];
    float f0 = -d_gzz[i];

    // Compute mean and determinant for eigenvalue calculation
    float m = (a0 + d0 + f0) / 3;
    float q = computeDeterminant(a0 - m, b0, c0, b0, d0 - m, e0, c0, e0, f0 - m) / 2;

    // Simplified computation of p using symmetry and removing redundancy
    float p = ((a0 - m) * (a0 - m) + 2 * b0 * b0 + 2 * c0 * c0 + (d0 - m) * (d0 - m) + 2 * e0 * e0 + (f0 - m) * (f0 - m)) / 6;

    float phi = 1.f / 3.f * atan(sqrt(p * p * p - q * q) / q);
    if (phi < 0) phi += 3.14159f / 3;

    float sqrt_p = sqrt(p); // Cache sqrt(p) for reuse
    float eig1 = m + 2 * sqrt_p * cos(phi);
    float eig2 = m - sqrt_p * (cos(phi) + sqrt(3.0f) * sin(phi));
    float eig3 = m - sqrt_p * (cos(phi) - sqrt(3.0f) * sin(phi));

    if (eig1 > eig2 && eig1 > eig3) d_output[i] = eig1 * sigma * sigma;
    else if (eig2 > eig1 && eig2 > eig3) d_output[i] = eig2 * sigma * sigma;
    else if (eig3 > eig2 && eig3 > eig1) d_output[i] = eig3 * sigma * sigma;

    // Eigenvector computation
    float l = d_output[i] / (sigma * sigma);
    a0 -= l; d0 -= l; f0 -= l;
    float xv = b0 * e0 - c0 * d0;
    float yv = e0 * a0 - c0 * b0;
    float zv = d0 * a0 - b0 * b0;
    float radius = sqrt(xv * xv + yv * yv + zv * zv);

    d_output_theta[i] = atan2(yv, xv);
    d_output_phi[i] = (radius > 1e-6f) ? acos(zv / radius) : 0;
}