#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTranspose(float* a, float* dest, int width, int height) {
    const int bx = blockIdx.x * blockDim.x;
    const int by = blockIdx.y * blockDim.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    __shared__ float smem[ADD_BLOCK_SIZE][ADD_BLOCK_SIZE + 1];

    int row = by + ty;
    int col = bx + tx;

    // Load input elements into shared memory
    if (col < width && row < height) {
        smem[ty][tx] = a[row * width + col];
    }
    __syncthreads();

    // Transpose the elements and store in destination matrix
    row = bx + ty;
    col = by + tx;

    if (row < width && col < height) {
        dest[col * width + row] = smem[tx][ty];
    }
}