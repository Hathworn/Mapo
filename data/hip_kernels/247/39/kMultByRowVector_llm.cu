#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate thread-specific row and column
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds to ensure no out-of-bounds memory access
    if (row < height && col < width) {
        // Flattened index in matrix
        unsigned int idx = row * width + col;
        // Perform multiplication
        tgtMat[idx] = mat[idx] * vec[col];
    }
}