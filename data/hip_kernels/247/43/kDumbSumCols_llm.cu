#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDumbSumCols(float* mat, float* vec, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < width) {
        float sum = 0;
        // Use a temporary pointer to avoid incrementing the original matrix pointer.
        float* matPtr = mat + idx; 
        for (int j = 0; j < height; j++) {
            sum += *matPtr;
            matPtr += width; // Move to the next element in the same column.
        }
        vec[idx] = sum;
    }
}