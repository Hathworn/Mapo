#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kInRangeInc(float* gData, float lower, float upper, float* target, unsigned int numElements) {
    // Use a local variable to reduce global memory access
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure idx is within numElements and use simpler loop unrolling to access memory less frequently
    if (idx < numElements) {
        for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x) {
            // Use local variables to reduce register pressure
            float value = gData[i];
            target[i] = (value >= lower) && (value <= upper);
        }
    }
}