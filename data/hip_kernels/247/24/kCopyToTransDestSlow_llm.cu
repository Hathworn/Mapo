#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline int getTransArrayIndex(unsigned int width, unsigned int height, unsigned int i) {
    return height * (i % width) + i / width;
}

__global__ void kCopyToTransDest(float* srcStart, float* destStart, unsigned int srcCopyWidth, unsigned int srcJumpWidth, unsigned int destJumpHeight, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure idx is within bounds before proceeding
    if (idx >= numElements) return;

    // Pre-compute indices to reduce recalculation
    unsigned int srcRow = idx / srcCopyWidth;
    unsigned int srcCol = idx % srcCopyWidth;
    unsigned int srcIndex = srcRow * srcJumpWidth + srcCol;
    unsigned int destIndex = getTransArrayIndex(srcCopyWidth, destJumpHeight, idx);

    // Directly copy data from source to destination
    destStart[destIndex] = srcStart[srcIndex];
}