#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMult(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total number of threads
    const unsigned int numThreads = blockDim.x * gridDim.x;
    
    // Optimize loop by unrolling
    for (unsigned int i = idx; i < numEls; i += numThreads) {
        dest[i] = a[i] * b[i];
        
        // Unroll loop for better performance
        if (i + numThreads < numEls) {
            dest[i + numThreads] = a[i + numThreads] * b[i + numThreads];
        }
    }
}