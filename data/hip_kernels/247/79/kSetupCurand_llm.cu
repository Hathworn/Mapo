#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Improved: Use grid stride loop to ensure all threads can initialize states.
    for (uint index = tidx; index < NUM_RND_THREADS_PER_BLOCK * gridDim.x; index += NUM_RND_THREADS_PER_BLOCK * blockDim.x) {
        hiprand_init(seed, index, 0, &state[index]);
    }
}
```
