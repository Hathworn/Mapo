#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd3(float* a, const float* b, const float* c, const unsigned int numEls, const float scaleA, const float scaleB, const float scaleC) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += stride) {
        a[i] = scaleA * a[i] + scaleB * b[i] + scaleC * c[i];
    }
}