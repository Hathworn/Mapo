#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState* state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Pre-calculate sequence number to minimize redundant calculations.
    unsigned long long sequence_number = seed + tidx;
    hiprand_init(seed, sequence_number, 0, &state[tidx]);
}