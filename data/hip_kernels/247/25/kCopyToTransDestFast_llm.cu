#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCopyToTransDestFast(float* srcStart, float* destStart, unsigned int srcCopyWidth, unsigned int srcCopyHeight, unsigned int srcJumpSize, unsigned int destJumpSize) {
    const unsigned int idxX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idxY = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds to avoid out-of-bounds memory access
    if (idxX < srcCopyWidth && idxY < srcCopyHeight) {
        unsigned int srcReadIdx = idxY * srcJumpSize + idxX;
        unsigned int destWriteIdx = idxX * destJumpSize + idxY;
        
        // Use padded shared memory to avoid bank conflicts
        __shared__ float smem[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE + 1];
        
        smem[threadIdx.y][threadIdx.x] = srcStart[srcReadIdx];
        __syncthreads();
        
        destStart[destWriteIdx] = smem[threadIdx.x][threadIdx.y];
    }
}