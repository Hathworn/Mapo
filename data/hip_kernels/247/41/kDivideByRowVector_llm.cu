#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivideByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Compute row and column indices for better cache usage
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int row = blockIdx.y * blockDim.y + threadIdx.y; row < height; row += blockDim.y * gridDim.y) {
        for (unsigned int i = col; i < width; i += stride) {
            tgtMat[row * width + i] = __fdividef(mat[row * width + i], vec[i]);
        }
    }
}