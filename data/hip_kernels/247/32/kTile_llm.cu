#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const int srcWidth, const int srcHeight, const int tgtWidth, const int tgtHeight) {
    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total number of threads
    const int numThreads = blockDim.x * gridDim.x;
    const int totalElements = tgtWidth * tgtHeight;

    // Loop with stride pattern to cover all elements
    for (unsigned int i = idx; i < totalElements; i += numThreads) {
        // Calculate y and x for the target matrix
        const int y = i / tgtWidth;
        const int x = i % tgtWidth;
        // Wrap y and x around the source matrix
        const int srcY = y % srcHeight;
        const int srcX = x % srcWidth;
        // Map source to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}