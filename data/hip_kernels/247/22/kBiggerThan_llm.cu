#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kBiggerThan(float* __restrict__ gMat1, float* __restrict__ gMat2, float* __restrict__ gMatTarget, unsigned int numElements) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Check bounds and perform computation
    if (idx < numElements) {
        gMatTarget[idx] = gMat1[idx] > gMat2[idx];
    }
}