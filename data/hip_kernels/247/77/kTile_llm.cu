#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate the unique global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    
    // Loop through the target array using a step size equal to the total number of threads
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Calculate the original source coordinates using modulo and division
        const uint srcY = (i / tgtWidth) % srcHeight;
        const uint srcX = (i % tgtWidth) % srcWidth;
        // Assign the value from the source to the target array
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}