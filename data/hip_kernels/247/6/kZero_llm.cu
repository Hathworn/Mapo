#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kZero(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride for better coalescing and efficiency
    const unsigned int stride = blockDim.x * gridDim.x; 

    for (unsigned int i = idx; i < numElements; i += stride) {
        target[i] = 0;
    }
}