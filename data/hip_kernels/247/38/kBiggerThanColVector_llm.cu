#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBiggerThanColVector(float* mat, float* vec, float* tgtMat, const int width, const int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Optimize loop unrolling for better warp execution and memory access efficiency
    for (unsigned int i = idx; i < width * height; i += numThreads) {
        unsigned int vecIndex = i / width;  // Calculate once and reuse
        tgtMat[i] = mat[i] > vec[vecIndex];
    }
}