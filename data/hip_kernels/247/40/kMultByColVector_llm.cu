#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate column index once
    const unsigned int column = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    for (unsigned int i = column; i < width * height; i += stride) {
        unsigned int row = i / width; // Divide i into row and column
        tgtMat[i] = mat[i] * vec[row];
    }
}