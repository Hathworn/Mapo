#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogistic2(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numElements) { // Check to prevent out-of-bound access
        target[idx] = 1 / (1 + expf(-gData[idx]));
    }
}