#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddScalar(float* gData, float scalar, float* target, unsigned int numElements) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;  // Calculate stride

    for (unsigned int i = idx; i < numElements; i += stride) {
        target[i] = scalar + gData[i];
    }
}