#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSmallerThanScalar(float* gData, float scalar, float* target, unsigned int numElements) {
    // Calculate index considering the whole grid
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure work is only done within bounds
    if (idx < numElements) {
        target[idx] = gData[idx] < scalar;
    }
}