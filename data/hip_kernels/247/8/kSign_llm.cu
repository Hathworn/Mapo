#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSign(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize by using a single if-else statement to reduce branching
    if (idx < numElements) {
        int sign = 0;
        if (gData[idx] > 0) sign = 1;
        else if (gData[idx] < 0) sign = -1;
        target[idx] = sign;
    }
}