#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSqrt(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numElements) { // Optimize: Check boundary condition early
        target[idx] = sqrtf(gData[idx]);
    }
}