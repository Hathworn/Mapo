#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBiggerThanScalar(float* gData, float scalar, float* target, unsigned int numElements) {
    // Calculate the global index for the thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (idx < numElements) {
        target[idx] = gData[idx] > scalar;
    }
}