#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kEqualsColVector(float* mat, float* vec, float* tgtMat, const int width, const int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    const unsigned int row = idx / width; // Calculate row index for vec
    const unsigned int col = idx % width; // Calculate column index

    // Use stride loop to handle threads across large matrices
    for (unsigned int i = idx; i < width * height; i += stride) {
        tgtMat[i] = mat[i] == vec[row];
    }
}