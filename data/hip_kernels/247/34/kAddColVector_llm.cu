#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddColVector(float* mat, float* vec, float* tgtMat, const unsigned int width, const unsigned int height, const float scaleVec) {
    // Use shared memory for vec[] to optimize memory access
    extern __shared__ float sharedVec[];

    // Each block loads a portion of the vector to shared memory
    unsigned int colIdx = threadIdx.x;
    if (colIdx < width) {
        sharedVec[colIdx] = vec[colIdx];
    }
    __syncthreads(); // Ensure all threads have completed the memory load

    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        // Use shared memory and avoid global memory access for vec
        tgtMat[i] = mat[i] + scaleVec * sharedVec[i % width];
    }
}