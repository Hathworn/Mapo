#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSubtractFromScalar(float* gData, float scalar, float* target, unsigned int numElements) {
    // Calculate global thread ID
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process threads within bounds
    if (idx < numElements) {
        target[idx] = scalar - gData[idx];
    }
}