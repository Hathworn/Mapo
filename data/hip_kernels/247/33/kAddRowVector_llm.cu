#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height, float scaleVec) {
    // Calculate row and column indices for better memory access pattern
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width) {
        tgtMat[row * width + col] = mat[row * width + col] + scaleVec * vec[col];
    }
}