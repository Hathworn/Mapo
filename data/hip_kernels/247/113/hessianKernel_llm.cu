#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float computeDeterminant (float e00, float e01, float e02, float e10, float e11, float e12, float e20, float e21, float e22)
{
    return e00*e11*e22 - e00*e12*e21 + e10*e21*e02 - e10*e01*e22 + e20*e01*e12 - e20*e11*e02;
}

__global__ void hessianKernel (float *d_output, const float *d_gxx, const float *d_gxy, const float *d_gxz, const float *d_gyy, const float *d_gyz, const float *d_gzz, float sigma, int imageW, int imageH, int imageD)
{
    int n_blocks_per_width = imageW / blockDim.x;
    int z = blockIdx.x / n_blocks_per_width; // Simplified integer division
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = (blockIdx.x % n_blocks_per_width) * blockDim.x + threadIdx.x; // Use modulus for calculation
    int i = z * imageW * imageH + y * imageW + x;

    float a0 = -d_gxx[i], b0 = -d_gxy[i], c0 = -d_gxz[i];
    float e0 = -d_gyy[i], f0 = -d_gyz[i], k0 = -d_gzz[i];

    float m = (a0 + e0 + k0) / 3;
    float q = computeDeterminant(a0-m, b0, c0, b0, e0-m, f0, c0, f0, k0-m) / 2;
    float p = (a0-m)*(a0-m) + 2*(b0*b0 + c0*c0 + f0*f0) + (e0-m)*(e0-m) + (k0-m)*(k0-m);
    p /= 6;
    float phi = 1.0f/3.0f * acos(clamp(q / sqrt(p*p*p), -1.0f, 1.0f));

    float sqrtP = sqrt(p);
    float eig1 = m + 2 * sqrtP * cos(phi);
    float eig2 = m - sqrtP * (cos(phi) + sqrt(3.0f) * sin(phi));
    float eig3 = m - sqrtP * (cos(phi) - sqrt(3.0f) * sin(phi));

    float maxEig = fmaxf(fmaxf(eig1, eig2), eig3); // Calculate max eigenvalue
    d_output[i] = maxEig * sigma * sigma;
}