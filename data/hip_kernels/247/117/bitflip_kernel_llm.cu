#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitflip_kernel(float* M, int height, int row, int n) {
    // Use shared memory and load coalesced for optimization
    extern __shared__ float shared_M[];
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    for (unsigned int i = idx; i < n; i += stride) {
        // Load data into shared memory for better memory access patterns
        shared_M[threadIdx.x] = M[i * height + row];
        __syncthreads();

        // Bitflipping the value
        shared_M[threadIdx.x] = 1 - shared_M[threadIdx.x];
        __syncthreads();

        // Write back to global memory
        M[i * height + row] = shared_M[threadIdx.x];
    }
}