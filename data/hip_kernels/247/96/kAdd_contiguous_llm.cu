#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Unroll the loop for better performance
    unsigned int i = idx;
    while (i < numEls) {
        dest[i] = a[i] + b[i];
        i += stride;
    }
}