#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogistic1(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for common data to improve speed (assuming blockDim.x fixed)
    __shared__ float sharedData[256];  // Adjust size based on blockDim.x if needed

    // Loop unrolling - process more than one element per iteration when possible
    for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x * 4) {
        if (i < numElements) {
            sharedData[threadIdx.x] = gData[i];
            target[i] = (1 + tanhf(sharedData[threadIdx.x] / 2)) / 2;
        }
        if (i + blockDim.x * gridDim.x < numElements) {
            sharedData[threadIdx.x] = gData[i + blockDim.x * gridDim.x];
            target[i + blockDim.x * gridDim.x] = (1 + tanhf(sharedData[threadIdx.x] / 2)) / 2;
        }
        if (i + 2 * blockDim.x * gridDim.x < numElements) {
            sharedData[threadIdx.x] = gData[i + 2 * blockDim.x * gridDim.x];
            target[i + 2 * blockDim.x * gridDim.x] = (1 + tanhf(sharedData[threadIdx.x] / 2)) / 2;
        }
        if (i + 3 * blockDim.x * gridDim.x < numElements) {
            sharedData[threadIdx.x] = gData[i + 3 * blockDim.x * gridDim.x];
            target[i + 3 * blockDim.x * gridDim.x] = (1 + tanhf(sharedData[threadIdx.x] / 2)) / 2;
        }
    }
}