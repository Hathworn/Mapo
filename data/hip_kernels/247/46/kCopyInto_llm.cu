#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCopyInto(float* images, float* targets, const int imgSize, const int paddingSize, const int numImages) {
    // Calculate unique image index
    const int imgIdx = blockIdx.y * gridDim.x + blockIdx.x;
    if (imgIdx < numImages) {
        const int targetSize = imgSize + 2 * paddingSize;
        images += imgIdx * imgSize * imgSize;
        targets += imgIdx * targetSize * targetSize + paddingSize * targetSize + paddingSize;

        // Optimize thread loop increment and synchronicity by using blockDim
        for (int y = threadIdx.y; y < imgSize; y += blockDim.y) {
            for (int x = threadIdx.x; x < imgSize; x += blockDim.x) {
                targets[y * targetSize + x] = images[y * imgSize + x];
            }
        }
    }
}