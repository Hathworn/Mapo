#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kEqualsRowVector(float* mat, float* vec, float* tgtMat, const int width, const int height) {
    // Calculate 2D indices to reduce modulo operation
    const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width) {  // Check bounds
        // Access matrix using row-major order
        const unsigned int idx = row * width + col;
        tgtMat[idx] = mat[idx] == vec[col];
    }
}