#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCopy(float* srcStart, float* destStart, const int copyWidth, const int srcJumpWidth, const int destJumpWidth, const int numElements) {
    // Calculate global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Use a more efficient loop by incrementing with stride
    for (int i = idx; i < numElements; i += stride) {
        int row = i / copyWidth;
        int col = i % copyWidth;
        
        // Access source and destination with computed indices
        destStart[row * destJumpWidth + col] = srcStart[row * srcJumpWidth + col];
    }
}