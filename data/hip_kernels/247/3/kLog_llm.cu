#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLog(float* gData, float* target, unsigned int numElements) {
    // Calculate global index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process all elements in stride of total threads
    for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x) {
        target[i] = logf(gData[i]); // Use logf for vector processing
    }
}