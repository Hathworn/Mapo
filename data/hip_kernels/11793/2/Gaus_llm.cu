#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define MINVAL 1e-7

__global__ void Gaus(double* Mtr, int Size, int i)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if index is valid for processing
    if (index > i && index < Size)
    {
        double particial = -Mtr[i * Size + index] / Mtr[i * Size + i];

        // Optimize loop: unroll for potential performance gain
        #pragma unroll
        for (int z = i; z < Size; z++)
        {
            Mtr[z * Size + index] += Mtr[z * Size + i] * particial;
        }
    }
}