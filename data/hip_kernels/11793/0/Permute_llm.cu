#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MINVAL 1e-7

__global__ void Permute(double* Dev_Mtr, int* i, int* k, int* Dev_size)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index < *Dev_size)
    {
        // Calculating the base index only once to save redundant calculations
        int baseIndex = index * (*Dev_size);
        double tmp = Dev_Mtr[baseIndex + (*i)];
        Dev_Mtr[baseIndex + (*i)] = Dev_Mtr[baseIndex + (*k)];
        Dev_Mtr[baseIndex + (*k)] = tmp;
    }
}