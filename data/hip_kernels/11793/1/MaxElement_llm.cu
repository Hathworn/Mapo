#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MINVAL 1e-7

__global__ void MaxElement(double* Mtr, int Size, int i, int* strnum)
{
    double MaxValue = Mtr[i * Size + i];
    int localStrnum = i;

    for(int k = i + 1; k < Size; k++) // Start loop from i+1
    {
        double currentVal = fabs(Mtr[i * Size + k]); // Cache value access
        if(currentVal > fabs(MaxValue))
        {
            localStrnum = k; // Use local variable for strnum
            MaxValue = Mtr[i * Size + k];
        }
    }

    *strnum = (fabs(MaxValue) < MINVAL) ? -1 : localStrnum; // Ternary operator for single assignment
}