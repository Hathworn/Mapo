#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    // Calculate overall thread index
    long ii = blockIdx.x * blockDim.x + threadIdx.x + 
              (blockIdx.y * blockDim.y + threadIdx.y) * (blockDim.x * gridDim.x);
    
    // Early exit if index is out of bounds
    if (ii >= no_elements) return;

    // Loop unrolling for better performance
    int ipidx_base = translate_idx_inv(ii, d1, d2, d3, scale_factor, 0, 0);
    int ipidx_offset = 1;

    // Pre-fetch gradOutput_data pointer for efficiency
    const float *output_ptr = gradOutput_data + ipidx_base;

    // Accumulate the sum using loop unrolling
    #pragma unroll
    for (int i = 0; i < scale_factor; i++) {
        #pragma unroll
        for (int j = 0; j < scale_factor; j++) {
            gradInput_data[ii] += *(output_ptr + i * d3 * scale_factor + (j * ipidx_offset));
        }
    }
}