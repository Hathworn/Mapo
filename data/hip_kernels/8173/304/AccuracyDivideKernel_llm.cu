#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load accuracy using a shared memory to reduce global memory access cost
    __shared__ float shared_accuracy;
    
    if (threadIdx.x == 0) {
        // Only one thread performs the division and writes back to global memory
        shared_accuracy = *accuracy / N;
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        *accuracy = shared_accuracy;
    }
}