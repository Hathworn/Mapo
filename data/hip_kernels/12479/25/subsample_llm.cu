#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Calculate offsets based on thread/block ID
    int o = blockIdx.x;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;

    // Select input/output plane; weights and biases
    float *local_output = output + o * output_w * output_h;
    float *local_input = input + o * input_w * input_h;
    float the_weight = weight[k];
    float the_bias = bias[k];

    // Optimize by using shared memory
    __shared__ float shared_input[32*32];  // Adjust size as needed
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Load into shared memory
    for (int yy = yy_start; yy < output_h; yy+=blockDim.y*gridDim.y) {
        for (int xx = xx_start; xx < output_w; xx+=blockDim.x) {
            int input_index = (yy * dH) * input_w + (xx * dW);
            shared_input[ty * blockDim.x + tx] = local_input[input_index];

            __syncthreads();  // Ensure all threads have written to shared memory

            float sum = 0;
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    sum += shared_input[(ty * dH + ky) * blockDim.x + (tx * dW + kx)];
                }
            }
            // Update output
            local_output[yy * output_w + xx] = the_weight * sum + the_bias;
        }
    }
}