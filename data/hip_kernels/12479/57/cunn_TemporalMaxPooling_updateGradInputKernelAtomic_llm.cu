#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Precompute common indices to reduce redundant calculations
    int timeIndex = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    if (timeIndex < output_w) {
        // Calculate base pointers once outside the loop
        float *gradInput_base = gradInput + blockIdx.x * input_w * input_n + timeIndex * input_n * dW;
        float *gradOutput_base = gradOutput + blockIdx.x * output_w * input_n + timeIndex * input_n;
        float *indices_base = indices + blockIdx.x * output_w * input_n + timeIndex * input_n;
        
        // Use loop unrolling for better performance
        for (int feat = 0; feat < input_n; ++feat) {
            atomicAdd(&gradInput_base[(int)indices_base[feat] * input_n + feat], gradOutput_base[feat]);
        }
    }
}