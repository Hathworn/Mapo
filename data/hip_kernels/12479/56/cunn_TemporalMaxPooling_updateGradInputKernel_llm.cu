#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 256  // Define block size for optimizations

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Optimize memory access by using local threadIdx variables
    int timeIndex = threadIdx.x + blockIdx.y * BLOCK_SIZE;

    if (timeIndex < output_w) {  // Ensure threads don't exceed output dimensions
        float *gradInput_data = gradInput + blockIdx.x * input_w * input_n + timeIndex * input_n * dW;
        float *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + timeIndex * input_n;
        float *indices_data = indices + blockIdx.x * output_w * input_n + timeIndex * input_n;

        // Loop through features using optimized access pattern
        for (int feat = 0; feat < input_n; ++feat) {
            int idx = (int)indices_data[feat] * input_n + feat;
            atomicAdd(&gradInput_data[idx], gradOutput_data[feat]);  // Use atomicAdd for safe parallel updates
        }
    }
}