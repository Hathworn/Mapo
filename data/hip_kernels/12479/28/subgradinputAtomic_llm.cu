#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // compute thread indices
    int xx = blockIdx.x * blockDim.x + threadIdx.x; 
    int yy = blockIdx.y * blockDim.y + threadIdx.y;

    // output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // bounds check
    if (xx >= output_w || yy >= output_h) return;

    // compute offsets
    int o = blockIdx.z;
    int i = o;
    int k = blockIdx.z % input_n;

    // select input/output plane
    gradOutput = gradOutput + o * output_w * output_h;
    gradInput = gradInput + i * input_w * input_h;

    // get weight
    float the_weight = weight[k];

    // compute pointers
    float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
    float *ptr_gradOutput = gradOutput + yy * output_w + xx;
    float z = *ptr_gradOutput * the_weight;

    // improve memory coalescing by ensuring atomic operations are contiguous
    for (int ky = 0; ky < kH; ky++)
    {
        for (int kx = 0; kx < kW; kx++)
        {
            atomicAdd(&ptr_gradInput[kx], z);
        }
        ptr_gradInput += input_w; // move to next row
    }
}