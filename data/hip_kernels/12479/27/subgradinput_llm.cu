#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradinput(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets
    int o = blockIdx.x;
    int k = o % input_n;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;
    int yy_start = blockIdx.y * blockDim.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Adjust pointers for the current block
    gradOutput += o * output_w * output_h;
    gradInput += o * input_w * input_h;

    // Preload weight
    float the_weight = weight[k];

    // Optimize kH and kW loop nesting by reducing index calculations
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        int input_row_offset = yy * dH * input_w;
        int output_row_offset = yy * output_w;
        
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            float *ptr_gradInput = gradInput + input_row_offset + xx * dW;
            float z = gradOutput[output_row_offset + xx] * the_weight;

            // Reduce index recalculations inside the loop
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    ptr_gradInput[kx] += z;
                }
                ptr_gradInput += input_w;
            }
        }
    }
}