#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

// Global kernel optimized for performance
__global__ void subtract(float *d_dst, const float *d_src_1, const float *d_src_2, int len) {
    // Calculate the global thread index
    int baseX = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index within array bounds
    if (baseX < len) {
        d_dst[baseX] = d_src_1[baseX] - d_src_2[baseX];
    }
}