#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void convolutionColumnsKernel_down_smp(float *d_Dst, float *d_Src, int imageW, int imageH, int n_imageH, int pitch, int filter_Rad, int Halo_steps)
{
    extern __shared__ float s_Data[];

    // Calculate base indices
    const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
    const int baseY = (blockIdx.y * 2 * COLUMNS_RESULT_STEPS - Halo_steps) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
    const int baseY1 = (blockIdx.y * COLUMNS_RESULT_STEPS) * COLUMNS_BLOCKDIM_Y + threadIdx.y;

    if (baseX < imageW)
    {
        d_Src += baseY * pitch + baseX;
        d_Dst += baseY1 * pitch + baseX;
        
        // Use shared memory more efficiently with correct indexing
        int shared_mem_offset = threadIdx.x * (2 * COLUMNS_RESULT_STEPS + 2 * Halo_steps) * COLUMNS_BLOCKDIM_Y + threadIdx.y;
        
        // Load upper halo
        #pragma unroll
        for (int i = 0; i < Halo_steps; i++)
        {
            s_Data[shared_mem_offset + i * COLUMNS_BLOCKDIM_Y] = (baseY + i * COLUMNS_BLOCKDIM_Y >= 0) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
        }
        
        // Load lower halo + main data
        #pragma unroll
        for (int i = Halo_steps; i < Halo_steps + 2 * COLUMNS_RESULT_STEPS + Halo_steps; i++)
        {
            s_Data[shared_mem_offset + i * COLUMNS_BLOCKDIM_Y] = (baseY + i * COLUMNS_BLOCKDIM_Y < imageH) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
        }
        
        // Ensure all data is loaded into shared memory
        __syncthreads();
        
        // Compute and store results
        #pragma unroll
        for (int i = 0; i < COLUMNS_RESULT_STEPS; ++i)
        {
            float sum = 0;
            if (baseY1 + i * COLUMNS_BLOCKDIM_Y < n_imageH)
            {
                #pragma unroll
                for (int j = -filter_Rad; j <= filter_Rad; ++j)
                {
                    sum += c_Kernel[filter_Rad - j] * s_Data[shared_mem_offset + 2 * threadIdx.y + 2 * i * COLUMNS_BLOCKDIM_Y + Halo_steps * COLUMNS_BLOCKDIM_Y + j];
                }
                d_Dst[i * COLUMNS_BLOCKDIM_Y * pitch] = sum;
            }
        }
    }
}