#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dwt_per_X(float *d_ip, int rows, int cols, int cA_cols, int filt_len, int Halo_steps, float *d_cL, float *d_cH)
{
    extern __shared__ float s_Data[];

    // Compute base offsets
    const int baseX = ((blockIdx.x * 2 * X_RESULT_STEPS) - Halo_steps) * X_BLOCKDIM_X + threadIdx.x;
    const int baseX1 = (blockIdx.x * X_RESULT_STEPS) * X_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * X_BLOCKDIM_Y + threadIdx.y;

    if (baseY < rows) {

        d_ip += baseY * cols + baseX;
        d_cL += baseY * cA_cols + baseX1;
        d_cH += baseY * cA_cols + baseX1;

        // Load data to shared memory with simplified bounds handling
        #pragma unroll
        for (int i = 0; i < Halo_steps; i++)
        {
            int index = baseX + i * X_BLOCKDIM_X;
            index = (index < 0) ? cols - 1 : index;  // Handle left boundary
            s_Data[(threadIdx.y * (2 * X_RESULT_STEPS + 2 * Halo_steps) * X_BLOCKDIM_X) + threadIdx.x + i * X_BLOCKDIM_X] = (index >= cols) ? d_ip[index - cols] : d_ip[index];
        }

        // Main data and Load right halo
        #pragma unroll
        for (int i = Halo_steps; i < Halo_steps + 2 * X_RESULT_STEPS + Halo_steps; i++)
        {
            int index = baseX + i * X_BLOCKDIM_X;
            index = (index >= cols) ? index - cols : index;  // Handle right boundary
            s_Data[(threadIdx.y * (2 * X_RESULT_STEPS + 2 * Halo_steps) * X_BLOCKDIM_X) + threadIdx.x + i * X_BLOCKDIM_X] = d_ip[index];
        }

        __syncthreads();

        // Compute results and store
        #pragma unroll
        for (int i = 0; i < X_RESULT_STEPS; i++)
        {
            if ((baseX1 + i * X_BLOCKDIM_X < cA_cols))
            {
                float sum_cL = 0, sum_cH = 0;
                int l2 = filt_len / 2;

                for (int l = 0; l < filt_len; ++l)
                {
                    int index = threadIdx.y * (2 * X_RESULT_STEPS + 2 * Halo_steps) * X_BLOCKDIM_X + threadIdx.x * 2 + Halo_steps * X_BLOCKDIM_X + 2 * i * X_BLOCKDIM_X + l2 - l;
                    sum_cL += c_lpd[l] * s_Data[index];
                    sum_cH += c_hpd[l] * s_Data[index];
                }
                d_cL[i * X_BLOCKDIM_X] = sum_cL;
                d_cH[i * X_BLOCKDIM_X] = sum_cH;
            }
        }
    }
}