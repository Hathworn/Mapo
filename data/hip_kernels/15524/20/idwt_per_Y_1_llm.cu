#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimization: Use more intuitive indexing and reduce arithmetic redundancy
__global__ void idwt_per_Y_1_optimized(float *d_dst, float *src_A, float *src_D, int rows, int cols, int next_rows, int filt_len, int halo) {

    extern __shared__ float s_Data[];

    // Offset to the upper halo edge
    int baseX = blockIdx.x * I_Y_BLOCKDIM_X + threadIdx.x;
    int baseY = ((blockIdx.y * I_Y_RESULT_STEPS) - halo) * I_Y_BLOCKDIM_Y + threadIdx.y;

    int l2 = filt_len / 2;

    if (baseX < cols) {
        src_A += baseY * cols + baseX;
        src_D += baseY * cols + baseX;
        d_dst += (2 * baseY - l2 + 1) * cols + baseX;

        int sharedIdx_A = threadIdx.x * (I_Y_RESULT_STEPS + 2 * halo) * I_Y_BLOCKDIM_Y;
        int sharedIdx_D = sharedIdx_A + I_Y_BLOCKDIM_X * (I_Y_RESULT_STEPS + 2 * halo) * I_Y_BLOCKDIM_Y;
        
        // Loading data to shared memory
        // Upper halo
        #pragma unroll
        for (int i = 0; i < halo; i++) {
            int yIdx = baseY + i * I_Y_BLOCKDIM_Y;
            int offset = i * I_Y_BLOCKDIM_Y * cols;
            s_Data[sharedIdx_A + threadIdx.y + i * I_Y_BLOCKDIM_Y] = (yIdx >= 0) ? src_A[offset] : src_A[offset + (rows * cols)];
            s_Data[sharedIdx_D + threadIdx.y + i * I_Y_BLOCKDIM_Y] = (yIdx >= 0) ? src_D[offset] : src_D[offset + (rows * cols)];
        }

        // Lower halo + Main data
        #pragma unroll
        for (int i = halo; i < halo + I_Y_RESULT_STEPS + halo; i++) {
            int yIdx = baseY + i * I_Y_BLOCKDIM_Y;
            int offset = i * I_Y_BLOCKDIM_Y * cols;
            s_Data[sharedIdx_A + threadIdx.y + i * I_Y_BLOCKDIM_Y] = (yIdx < rows) ? src_A[offset] : src_A[offset - (rows * cols)];
            s_Data[sharedIdx_D + threadIdx.y + i * I_Y_BLOCKDIM_Y] = (yIdx < rows) ? src_D[offset] : src_D[offset - (rows * cols)];
        }

        __syncthreads();
        
        // Calculation
        #pragma unroll
        for (int i = halo; i < I_Y_RESULT_STEPS + halo; i++) {
            int pos_y = 2 * baseY + 2 * i * I_Y_BLOCKDIM_Y;

            if (pos_y + 1 < (2 * rows + filt_len - 2)) {
                float temp_1 = 0, temp_2 = 0;

                #pragma unroll
                for (int l = 0; l < l2; ++l) {
                    int t = 2 * l;
                    int dataIdx = threadIdx.y + i * I_Y_BLOCKDIM_Y - l;
                    
                    temp_1 += c_lpr[t] * s_Data[sharedIdx_A + dataIdx] + c_hpr[t] * s_Data[sharedIdx_D + dataIdx];
                    temp_2 += c_lpr[t + 1] * s_Data[sharedIdx_A + dataIdx] + c_hpr[t + 1] * s_Data[sharedIdx_D + dataIdx];
                }

                if ((pos_y >= l2 - 1) && (pos_y < next_rows + l2 - 1)) {
                    d_dst[2 * i * I_Y_BLOCKDIM_Y * cols] = temp_1;
                }
                if ((pos_y + 1 >= l2 - 1) && (pos_y + 1 < next_rows + l2 - 1)) {
                    d_dst[(2 * i * I_Y_BLOCKDIM_Y + 1) * cols] = temp_2;
                }
            }
        }
    }
}