#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void add(float *d_dst, const float *d_src_1, int len) {
    int baseX = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for better performance in future complex operations
    if (baseX < len) {
        d_dst[baseX] += d_src_1[baseX]; // Simplified in-place addition
    }
}