#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dwt_per_Y_O(float *d_ip, int rows, int cols, int cA_rows, int filt_len, int Halo_steps, float *d_cL, float *d_cH)
{
    extern __shared__ float s_Data[];

    // Calculate global and shared memory indices
    const int baseX = blockIdx.x * Y_BLOCKDIM_X + threadIdx.x;
    const int baseY = ((blockIdx.y * 2 * Y_RESULT_STEPS) - Halo_steps) * Y_BLOCKDIM_Y + threadIdx.y;
    const int baseY1 = (blockIdx.y * Y_RESULT_STEPS) * Y_BLOCKDIM_Y + threadIdx.y;
    const int sharedStride = (2 * Y_RESULT_STEPS + 2 * Halo_steps) * Y_BLOCKDIM_Y;
    float *s_Data_current = s_Data + threadIdx.x * sharedStride + threadIdx.y;
    
    if (baseX < cols)
    {
        // Update input pointers to point to the specific section of the array
        d_ip += baseY * cols + baseX;
        d_cL += baseY1 * cols + baseX;
        d_cH += baseY1 * cols + baseX;
        
        // Load data to shared memory, reduce conditions and bounds checks
        #pragma unroll
        for (int i = 0; i < (Halo_steps + 2 * Y_RESULT_STEPS + Halo_steps); i++)
        {
            int global_y = baseY + i * Y_BLOCKDIM_Y;
            if (global_y >= 0 && global_y < rows) 
            {
                s_Data_current[i * Y_BLOCKDIM_Y] = d_ip[i * Y_BLOCKDIM_Y * cols];
            }
            else if (global_y < 0)
            {
                s_Data_current[i * Y_BLOCKDIM_Y] = d_ip[(rows - 1) * cols];
            }
            else
            {
                s_Data_current[i * Y_BLOCKDIM_Y] = d_ip[(rows - 1) * cols];
            }
        }
        __syncthreads();
        
        // Compute results and store them
        #pragma unroll
        for (int i = 0; i < Y_RESULT_STEPS; i++)
        {
            if ((baseY1 + i * Y_BLOCKDIM_Y) < cA_rows)
            {
                float sum_cL = 0, sum_cH = 0;
                int offset = 2 * threadIdx.y + 2 * i * Y_BLOCKDIM_Y + Halo_steps * Y_BLOCKDIM_Y + (filt_len / 2);

                // Filter application using simple arithmetic operations
                #pragma unroll
                for (int l = 0; l < filt_len; ++l)
                {
                    sum_cL += c_lpd[l] * s_Data_current[offset - l];
                    sum_cH += c_hpd[l] * s_Data_current[offset - l];
                }
                d_cL[i * Y_BLOCKDIM_Y * cols] = sum_cL;
                d_cH[i * Y_BLOCKDIM_Y * cols] = sum_cH;
            }
        }
    }
}