#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void dwt_compare(float *d_ip_v, float *d_ip_ir, int len) {
    const int X = blockIdx.x * blockDim.x + threadIdx.x;
    if (X < len) {
        float abs_v = fabsf(d_ip_v[X]);  // Use fabsf for single-precision floats
        float abs_ir = fabsf(d_ip_ir[X]); // Use fabsf for single-precision floats
        d_ip_v[X] = (abs_v > abs_ir) ? d_ip_v[X] : d_ip_ir[X];
    }
}