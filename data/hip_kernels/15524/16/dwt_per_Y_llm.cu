#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dwt_per_Y(float *d_ip, int rows, int cols, int cA_rows, int filt_len, int Halo_steps, float *d_cL, float *d_cH) {

    extern __shared__ float s_Data[];

    // Offset to the upper halo edge
    const int baseX = blockIdx.x * Y_BLOCKDIM_X + threadIdx.x;
    const int baseY = ((blockIdx.y * 2 * Y_RESULT_STEPS) - Halo_steps) * Y_BLOCKDIM_Y + threadIdx.y;
    const int baseY1 = (blockIdx.y * Y_RESULT_STEPS) * Y_BLOCKDIM_Y + threadIdx.y;

    if (baseX < cols) {
        d_ip += baseY * cols + baseX;
        d_cL += baseY1 * cols + baseX;
        d_cH += baseY1 * cols + baseX;

        // Precompute indices for shared memory access
        const int sDataOffset = threadIdx.x * (2 * Y_RESULT_STEPS + 2 * Halo_steps) * Y_BLOCKDIM_Y + threadIdx.y;

        // Loading data to shared memory
        if (rows % 2 == 1) {
            // Upper halo
            #pragma unroll
            for (int i = 0; i < Halo_steps; i++) {
                int offset = sDataOffset + i * Y_BLOCKDIM_Y;
                if (baseY + i * Y_BLOCKDIM_Y == -1)
                    s_Data[offset] = d_ip[(rows - 1) * cols];
                else
                    s_Data[offset] = (baseY >= -i * Y_BLOCKDIM_Y) ? 
                        d_ip[i * Y_BLOCKDIM_Y * cols] : 
                        d_ip[(i * Y_BLOCKDIM_Y * cols) + ((rows + 1) * cols)];
            }

            // Lower halo + Main data
            #pragma unroll
            for (int i = Halo_steps; i < Halo_steps + 2 * Y_RESULT_STEPS + Halo_steps; i++) {
                int offset = sDataOffset + i * Y_BLOCKDIM_Y;
                if (baseY + i * Y_BLOCKDIM_Y == rows)
                    s_Data[offset] = d_ip[(i * Y_BLOCKDIM_Y * (cols - 1))];
                else
                    s_Data[offset] = (rows - baseY > i * Y_BLOCKDIM_Y) ? 
                        d_ip[i * Y_BLOCKDIM_Y * cols] : 
                        d_ip[(i * Y_BLOCKDIM_Y * cols) - ((rows + 1) * cols)];
            }
        } else {
            // Upper halo
            #pragma unroll
            for (int i = 0; i < Halo_steps; i++) {
                int offset = sDataOffset + i * Y_BLOCKDIM_Y;
                s_Data[offset] = (baseY >= -i * Y_BLOCKDIM_Y) ? 
                    d_ip[i * Y_BLOCKDIM_Y * cols] : 
                    d_ip[(i * Y_BLOCKDIM_Y * cols) + (rows * cols)];
            }

            // Lower halo + Main data
            #pragma unroll
            for (int i = Halo_steps; i < Halo_steps + 2 * Y_RESULT_STEPS + Halo_steps; i++) {
                int offset = sDataOffset + i * Y_BLOCKDIM_Y;
                s_Data[offset] = (rows - baseY > i * Y_BLOCKDIM_Y) ? 
                    d_ip[i * Y_BLOCKDIM_Y * cols] : 
                    d_ip[(i * Y_BLOCKDIM_Y * cols) - (rows * cols)];
            }
        }

        __syncthreads();

        // Compute and store results
        #pragma unroll
        for (int i = 0; i < Y_RESULT_STEPS; i++) {
            if ((baseY1 + i * Y_BLOCKDIM_Y < cA_rows)) {
                int l2 = filt_len / 2;

                float sum_cL = 0, sum_cH = 0;

                #pragma unroll
                for (int l = 0; l < filt_len; ++l) {
                    int index = sDataOffset + 2 * threadIdx.y + 2 * i * Y_BLOCKDIM_Y + Halo_steps * Y_BLOCKDIM_Y + l2 - l;
                    sum_cL += c_lpd[l] * s_Data[index];
                    sum_cH += c_hpd[l] * s_Data[index];
                }

                d_cL[i * Y_BLOCKDIM_Y * cols] = sum_cL;
                d_cH[i * Y_BLOCKDIM_Y * cols] = sum_cH;
            }
        }
    }
}