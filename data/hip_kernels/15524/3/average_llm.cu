#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void average(float *d_ip_v, float *d_ip_ir, int app_len) {

    // Calculate the global thread index
    int X = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce global memory access
    extern __shared__ float shared_mem[];
    float *s_ip_v = shared_mem;
    float *s_ip_ir = shared_mem + blockDim.x;

    // Load inputs into shared memory
    if (X < app_len) {
        s_ip_v[threadIdx.x] = d_ip_v[X];
        s_ip_ir[threadIdx.x] = d_ip_ir[X];
    }
    __syncthreads();

    // Calculate the average using shared memory
    if (X < app_len) {
        s_ip_v[threadIdx.x] = (s_ip_v[threadIdx.x] + s_ip_ir[threadIdx.x]) / 2;
        d_ip_v[X] = s_ip_v[threadIdx.x];
    }
}