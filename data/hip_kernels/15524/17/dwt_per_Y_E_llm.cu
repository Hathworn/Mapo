#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dwt_per_Y_E(float *d_ip, int rows, int cols, int cA_rows, int filt_len, int Halo_steps, float *d_cL, float *d_cH) {
    extern __shared__ float s_Data[];

    // Calculate base indices
    const int baseX = blockIdx.x * Y_BLOCKDIM_X + threadIdx.x;
    const int baseY = ((blockIdx.y * 2 * Y_RESULT_STEPS) - Halo_steps) * Y_BLOCKDIM_Y + threadIdx.y;
    const int baseY1 = (blockIdx.y * Y_RESULT_STEPS) * Y_BLOCKDIM_Y + threadIdx.y;

    if (baseX < cols) {
        d_ip += baseY * cols + baseX;
        d_cL += baseY1 * cols + baseX;
        d_cH += baseY1 * cols + baseX;

        // Load data to shared memory with boundary check
        #pragma unroll
        for (int i = 0; i < Halo_steps + 2 * Y_RESULT_STEPS + Halo_steps; i++) {
            int globalY = baseY + i * Y_BLOCKDIM_Y;
            int sharedIdx = (threadIdx.x * (2 * Y_RESULT_STEPS + 2 * Halo_steps) * Y_BLOCKDIM_Y) + threadIdx.y + i * Y_BLOCKDIM_Y;
            s_Data[sharedIdx] = (globalY >= 0 && globalY < rows) ? d_ip[i * Y_BLOCKDIM_Y * cols] : 0.0f;
        }

        __syncthreads();

        // Compute and store results
        #pragma unroll
        for (int i = 0; i < Y_RESULT_STEPS; i++) {
            if ((baseY1 + i * Y_BLOCKDIM_Y) < cA_rows) {
                int l2 = filt_len / 2;
                float sum_cL = 0.0f, sum_cH = 0.0f;
                
                #pragma unroll
                for (int l = 0; l < filt_len; ++l) {
                    int sharedIdx = (threadIdx.x * (2 * Y_RESULT_STEPS + 2 * Halo_steps) * Y_BLOCKDIM_Y) + 2 * threadIdx.y + 2 * i * Y_BLOCKDIM_Y + Halo_steps * Y_BLOCKDIM_Y + l2 - l;
                    sum_cL += c_lpd[l] * s_Data[sharedIdx];
                    sum_cH += c_hpd[l] * s_Data[sharedIdx];
                }

                d_cL[i * Y_BLOCKDIM_Y * cols] = sum_cL;
                d_cH[i * Y_BLOCKDIM_Y * cols] = sum_cH;
            }
        }
    }
}