#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void idwt_per_X_1(float *d_dst, float *src_A, float *src_D, int rows, int cols, int next_cols, int filt_len, int halo) {
    extern __shared__ float s_Data[];

    // Offset to the left halo edge
    const int baseX = ((blockIdx.x * I_X_RESULT_STEPS) - halo) * I_X_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * I_X_BLOCKDIM_Y + threadIdx.y;
    const int baseX1 = blockIdx.x * I_X_RESULT_STEPS * 2 * I_X_BLOCKDIM_X + 2 * threadIdx.x;

    if (baseY < rows) {
        src_A += baseY * cols + baseX;
        src_D += baseY * cols + baseX;
        d_dst += baseY * next_cols + baseX1;

        int l2 = filt_len / 2;

        // Loading data to shared memory, with boundary checks condensed
        #pragma unroll
        for (int i = -halo; i < halo + I_X_RESULT_STEPS + halo; ++i) {
            int src_index = baseX + i * I_X_BLOCKDIM_X;

            // Compute clamped indices for loading
            int src_index_A = max(0, min(src_index, cols - 1));
            int src_index_D = max(0, min(src_index, cols - 1));

            s_Data[(threadIdx.y * (I_X_RESULT_STEPS + 2 * halo) * I_X_BLOCKDIM_X) + threadIdx.x + (i + halo) * I_X_BLOCKDIM_X] = src_A[src_index_A];
            s_Data[((threadIdx.y + I_X_BLOCKDIM_Y) * (I_X_RESULT_STEPS + 2 * halo) * I_X_BLOCKDIM_X) + threadIdx.x + (i + halo) * I_X_BLOCKDIM_X] = src_D[src_index_D];
        }

        // Compute and store results
        __syncthreads();

        #pragma unroll
        for (int i = halo; i < halo + I_X_RESULT_STEPS; ++i) {
            int pos_x = (baseX1 + 2 * (i - halo) * I_X_BLOCKDIM_X);

            if ((pos_x + 1) < (2 * cols + filt_len - 2)) {
                float temp_1 = 0, temp_2 = 0;

                // Ensure memory access patterns are contiguous
                #pragma unroll
                for (int l = 0; l < l2; ++l) {
                    int t = 2 * l;

                    float val_A1 = s_Data[(threadIdx.y * (I_X_RESULT_STEPS + 2 * halo) * I_X_BLOCKDIM_X) + threadIdx.x + i * I_X_BLOCKDIM_X - l];
                    float val_D1 = s_Data[((threadIdx.y + I_X_BLOCKDIM_Y) * (I_X_RESULT_STEPS + 2 * halo) * I_X_BLOCKDIM_X) + threadIdx.x + i * I_X_BLOCKDIM_X - l];

                    temp_1 += c_lpr[t] * val_A1 + c_hpr[t] * val_D1;
                    temp_2 += c_lpr[t + 1] * val_A1 + c_hpr[t + 1] * val_D1;
                }

                if ((pos_x >= l2 - 1) && (pos_x < (next_cols + l2 - 1))) 
                    d_dst[2 * (i - halo) * I_X_BLOCKDIM_X - l2 + 1] = temp_1;
                if ((pos_x + 1 >= l2 - 1) && (pos_x + 1 < (next_cols + l2 - 1))) 
                    d_dst[2 * (i - halo) * I_X_BLOCKDIM_X - l2 + 2] = temp_2;
            }
        }
    }
}