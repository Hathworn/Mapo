#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void d_transpose(float *odata, float *idata, int width, int height)
{
    // Use shared memory for coalesced memory access
    __shared__ float block[BLOCK_DIM][BLOCK_DIM + 1];

    // Calculate input and output indices
    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;

    // Load data from global memory to shared memory
    if ((xIndex < width) && (yIndex < height))
    {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }

    __syncthreads();

    // Calculate new transposed indices
    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;

    // Store the transposed data from shared to global memory
    if ((xIndex < height) && (yIndex < width))
    {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}