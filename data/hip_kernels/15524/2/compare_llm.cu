#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void compare(float *d_ip_v, float *d_ip_ir, int len) {
    // Calculate global thread index
    const int X = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize with loop unrolling for potential performance gain
    #pragma unroll
    for (int i = X; i < len; i += gridDim.x * blockDim.x) {
        d_ip_v[i] = (abs(d_ip_v[i]) > abs(d_ip_ir[i])) ? d_ip_v[i] : d_ip_ir[i];
    }
}