#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dwt_average(float *d_ip_v, float *d_ip_ir, int app_len) {
    // Calculate index only for threads within bounds
    const int X = blockIdx.x * blockDim.x + threadIdx.x;
    if (X >= app_len) return; // Early exit for threads out of bounds

    // Perform the operation
    d_ip_v[X] = (d_ip_v[X] + d_ip_ir[X]) * 0.5f; // Use multiplication instead of division
}