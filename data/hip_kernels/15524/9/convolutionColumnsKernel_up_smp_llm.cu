#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float *c_Kernel;

__global__ void convolutionColumnsKernel_up_smp(float *d_Dst, float *d_Src, int imageW, int imageH, int n_imageH, int pitch, int filter_Rad, int Halo_steps) {
    extern __shared__ float s_Data[];

    // Calculate global indices for memory access
    const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
    const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - Halo_steps) * COLUMNS_BLOCKDIM_Y + threadIdx.y;

    if (baseX < imageW) {
        d_Src += baseY * pitch + baseX;
        d_Dst += 2 * baseY * pitch + baseX;

        // Load Upper and Lower halo and Main data
        #pragma unroll
        for (int i = 0; i < Halo_steps + COLUMNS_RESULT_STEPS + Halo_steps; i++) {
            int globalY = baseY + i * COLUMNS_BLOCKDIM_Y;
            // Combined halo and main data loading using a conditional operator
            s_Data[threadIdx.x * (COLUMNS_RESULT_STEPS + 2 * Halo_steps) * COLUMNS_BLOCKDIM_Y + threadIdx.y + i * COLUMNS_BLOCKDIM_Y] =
                (globalY >= 0 && globalY < imageH) ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch] : 0;
        }

        __syncthreads();  // Ensure all shared memory loads are completed

        // Compute and store results
        #pragma unroll
        for (int i = Halo_steps; i < COLUMNS_RESULT_STEPS + Halo_steps; ++i) {
            int Pos_y = 2 * baseY + (2 * i) * COLUMNS_BLOCKDIM_Y;

            if (Pos_y < n_imageH) {
                float sum_1 = 0.0f, sum_2 = 0.0f;

                // Unrolling filter radius loop for performance improvement
                #pragma unroll
                for (int l = -(filter_Rad / 2); l <= filter_Rad / 2; ++l) {
                    int t = 2 * l;
                    float temp = s_Data[threadIdx.x * (COLUMNS_RESULT_STEPS + 2 * Halo_steps) * COLUMNS_BLOCKDIM_Y +
                                        threadIdx.y + i * COLUMNS_BLOCKDIM_Y + l];

                    sum_1 += c_Kernel[filter_Rad + t] * temp * 2.0f;
                    sum_2 += c_Kernel[filter_Rad + t - 1] * temp * 2.0f;
                }

                sum_2 += c_Kernel[2 * filter_Rad] * 2.0f * 
                         s_Data[threadIdx.x * (COLUMNS_RESULT_STEPS + 2 * Halo_steps) * COLUMNS_BLOCKDIM_Y + 
                                threadIdx.y + i * COLUMNS_BLOCKDIM_Y + filter_Rad / 2 + 1];

                d_Dst[2 * i * COLUMNS_BLOCKDIM_Y * pitch] = sum_1;
                if (Pos_y + 1 < n_imageH) {
                    d_Dst[2 * i * COLUMNS_BLOCKDIM_Y * pitch + pitch] = sum_2;
                }
            }
        }
    }
}