#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define L2HYS_EPSILON 		0.01f
#define L2HYS_EPSILONHYS	1.0f
#define L2HYS_CLIP			0.2f
#define data_h2y            30

// Optimized kernel function
__global__ void smooth(float *in, float *out)
{
    int k = blockIdx.x;  // Use blockIdx for iteration over k
    int j = blockIdx.y;  // Use blockIdx for iteration over j
    int m_nBIN = 10;
    float *m_pCellFeatures = in;
    int t_nLineWidth = 70;
    float t_pTemp[10];

    // Process i within threads
    int i = threadIdx.x;
    if (i < m_nBIN) {
        int t_nLeft = (i - 1 + m_nBIN) % m_nBIN;
        int t_nRight = (i + 1) % m_nBIN;

        // Use registers for accessing elements
        float center = m_pCellFeatures[k * t_nLineWidth + j * m_nBIN + i];
        float left = m_pCellFeatures[k * t_nLineWidth + j * m_nBIN + t_nLeft];
        float right = m_pCellFeatures[k * t_nLineWidth + j * m_nBIN + t_nRight];

        t_pTemp[i] = center * 0.8f + left * 0.1f + right * 0.1f;
    }

    __syncthreads();

    if (i < m_nBIN)
    {
        out[k * t_nLineWidth + j * m_nBIN + i] = t_pTemp[i];
    }
}