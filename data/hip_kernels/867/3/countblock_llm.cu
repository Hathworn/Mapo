#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define L2HYS_EPSILON       0.01f
#define L2HYS_EPSILONHYS    1.0f
#define L2HYS_CLIP          0.2f
#define data_h2y            30

__global__ void countblock(float *in, float *out)
{
    // Calculate indexes using 2D thread and block indices
    unsigned int idxBlock = 70 * blockIdx.x;
    unsigned int idxThreadIn = (blockIdx.y + threadIdx.x) * 10;
    unsigned int idxThreadOut = 120 * blockIdx.x + 30 * blockIdx.y + 10 * threadIdx.x;

    float *ptr_in = in + idxBlock + idxThreadIn;  // Input pointer calculation
    float *ptr_out = out + idxThreadOut;          // Output pointer calculation

    ptr_out[threadIdx.y] = ptr_in[threadIdx.y];    // Parallel writing using y-dimension threads
}