#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define L2HYS_EPSILON       0.01f
#define L2HYS_EPSILONHYS    1.0f
#define L2HYS_CLIP          0.2f
#define data_h2y            30

__global__ void normalizeL2Hys(float *in,float *out)
{
    // Use shared memory for partial sum and reduce access latency
    __shared__ float partialSum[30];

    int bid = blockIdx.x;
    int tid = threadIdx.x;
    
    // Load input data into register to reduce global memory access
    float val = in[bid * 30 + tid];

    // Perform sum of squares in parallel
    partialSum[tid] = val * val;
    __syncthreads();

    // Use first thread to sum all partial results
    if (tid == 0) {
        float sum = 0;
        for (int i = 0; i < 30; ++i) {
            sum += partialSum[i];
        }
        partialSum[0] = rsqrt(sum) + L2HYS_EPSILONHYS * 30;
    }
    __syncthreads();
    
    // Apply normalization
    out[bid * 30 + tid] = val * (1.0f / partialSum[0]);
}