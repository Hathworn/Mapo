#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void myhistKernel(unsigned char * buffer,unsigned int * histo)
{
    __shared__ unsigned int temp[256];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // Initialize shared memory only for relevant threads
    if (threadIdx.x < 256) {
        temp[threadIdx.x] = 0;
    }
    __syncthreads();

    // Check the range before atomicAdd in shared memory
    if (offset < blockDim.x * blockDim.y * gridDim.x * gridDim.y) {
        atomicAdd(&temp[buffer[offset]], 1);
    }
    __syncthreads();

    // Use only 256 threads for the final write-back to global memory
    if (threadIdx.x < 256) {
        atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
    }
}