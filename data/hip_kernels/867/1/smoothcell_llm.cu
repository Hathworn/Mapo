#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define L2HYS_EPSILON       0.01f
#define L2HYS_EPSILONHYS    1.0f
#define L2HYS_CLIP          0.2f
#define data_h2y            30

__global__ void smoothcell(float *in, float *out) {
    int t_nleft = (threadIdx.x + 9) % 10; // Optimize modulo calculation
    int t_nright = (threadIdx.x + 1) % 10; // Simplify right neighbor calculation
    float *t_ptemp = in + blockIdx.x * 70 + blockIdx.y * 10;
    
    // Precompute and store values in shared memory
    __shared__ float shared_temp[10];
    shared_temp[threadIdx.x] = t_ptemp[threadIdx.x];
    __syncthreads();

    // Apply smoothing filter
    t_ptemp[threadIdx.x] = shared_temp[threadIdx.x] * 0.8f 
                         + 0.1f * shared_temp[t_nleft] 
                         + 0.1f * shared_temp[t_nright];
    __syncthreads();
    
    // Write output
    out[blockIdx.x * 70 + blockIdx.y * 10 + threadIdx.x] = t_ptemp[threadIdx.x];
    __syncthreads();
}