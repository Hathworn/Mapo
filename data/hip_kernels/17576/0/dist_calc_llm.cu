#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dist_calc(float *coord, int *close_num, int *close_flag, int *close_idx, int num_atom, int num_atom2) {
    // close_flag is a 1024 x num_atom2 int matrix initialized to 0.
    // close_idx: A num_atom x 200 int matrix, row i of which only the first close_num[i] elements are defined. (Otherwise it's -1).
    __shared__ float x_ref, y_ref, z_ref;
    __shared__ int temp[2048];

    // Calc distance
    for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
        if (threadIdx.x == 0) {
            x_ref = coord[3 * ii];
            y_ref = coord[3 * ii + 1];
            z_ref = coord[3 * ii + 2];
        }
        int idy = ii % gridDim.x; // Determine row of close_flag this block updates
        __syncthreads();

        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            float r2t = (coord[3 * jj] - x_ref) * (coord[3 * jj] - x_ref) +
                        (coord[3 * jj + 1] - y_ref) * (coord[3 * jj + 1] - y_ref) +
                        (coord[3 * jj + 2] - z_ref) * (coord[3 * jj + 2] - z_ref);

            if (r2t < 34.0) {
                close_flag[idy * num_atom2 + jj] = 1; // roughly 2 A + 2 A vdW + 2 * 1.8 A probe
            } else {
                close_flag[idy * num_atom2 + jj] = 0;
            }
            if (ii == jj) close_flag[idy * num_atom2 + jj] = 0;
        }
        __syncthreads();

        // Pre scan with optimized memory access
        int temp_sum = close_num[ii];
        int offset = 1;
        
        int idx = threadIdx.x;
        int idx2 = idx * 2;

        for (int jj = idx2; jj < num_atom2; jj += blockDim.x * 2) {
            temp[idx2] = close_flag[idy * num_atom2 + jj];
            temp[idx2 + 1] = close_flag[idy * num_atom2 + jj + 1];
        }
        
        for (int d = blockDim.x; d > 0; d >>= 1) { // up-sweep
            __syncthreads();
            if (idx < d) {
                int ai = offset * (idx2 + 1) - 1;
                int bi = offset * (idx2 + 2) - 1;
                temp[bi] += temp[ai];
            }
            offset *= 2;
        }
        
        __syncthreads();

        if (idx == 0) {
            close_num[ii] += temp[2 * blockDim.x - 1];
            temp[2 * blockDim.x - 1] = 0;
        }

        offset >>= 1;

        for (int d = 1; d < blockDim.x * 2; d *= 2) { // down-sweep
            __syncthreads();
            if (idx < d) {
                int ai = offset * (idx2 + 1) - 1;
                int bi = offset * (idx2 + 2) - 1;
                int t = temp[ai];
                temp[ai] = temp[bi];
                temp[bi] += t;
            }
            offset >>= 1;
        }

        __syncthreads();

        // Finally assign the indices
        if (close_flag[idy * num_atom2 + idx2] == 1) {
            close_idx[ii * 1024 + temp[idx2] + temp_sum] = idx2;
        }
        if (close_flag[idy * num_atom2 + idx2 + 1] == 1) {
            close_idx[ii * 1024 + temp[idx2 + 1] + temp_sum] = idx2 + 1;
        }
    }
}