#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void force_calc_EMA(float *Force, double *Force_old, int num_atom, int num_q, float *f_ptxc, float *f_ptyc, float *f_ptzc, int num_atom2, int num_q2, int *Ele, double EMA_norm, float force_ramp) {
    if (blockIdx.x >= num_atom) return;

    for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
        // Optimize stride loop using shared memory
        extern __shared__ float shared_ptx[], shared_pty[], shared_ptz[];
        for (int iAccum = threadIdx.x; iAccum < num_q2; iAccum += blockDim.x) {
            shared_ptx[iAccum] = f_ptxc[ii + iAccum * num_atom2];
            shared_pty[iAccum] = f_ptyc[ii + iAccum * num_atom2];
            shared_ptz[iAccum] = f_ptzc[ii + iAccum * num_atom2];
        }
        __syncthreads();

        for (int stride = num_q2 / 2; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                shared_ptx[threadIdx.x] += shared_ptx[threadIdx.x + stride];
                shared_pty[threadIdx.x] += shared_pty[threadIdx.x + stride];
                shared_ptz[threadIdx.x] += shared_ptz[threadIdx.x + stride];
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            if (Ele[ii]) {
                // Use atomic operations to avoid race conditions
                atomicAdd(&Force_old[ii * 3], (EMA_norm - 1.0) * Force_old[ii * 3] - (double)shared_ptx[0]);
                atomicAdd(&Force_old[ii * 3 + 1], (EMA_norm - 1.0) * Force_old[ii * 3 + 1] - (double)shared_pty[0]);
                atomicAdd(&Force_old[ii * 3 + 2], (EMA_norm - 1.0) * Force_old[ii * 3 + 2] - (double)shared_ptz[0]);

                Force[ii * 3] = (float)(Force_old[ii * 3] / EMA_norm) * force_ramp;
                Force[ii * 3 + 1] = (float)(Force_old[ii * 3 + 1] / EMA_norm) * force_ramp;
                Force[ii * 3 + 2] = (float)(Force_old[ii * 3 + 2] / EMA_norm) * force_ramp;
            }
        }
        __syncthreads();
    }
}