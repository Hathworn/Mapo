#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void force_calc(float *Force, int num_atom, int num_q, float *f_ptxc, float *f_ptyc, float *f_ptzc, int num_atom2, int num_q2, int *Ele, float force_ramp) {
    // Exit if block index is out of bounds
    if (blockIdx.x >= num_atom) return;

    // Iterate over atoms assigned to each block
    for (int ii = blockIdx.x; ii < num_atom; ii += gridDim.x) {
        // Perform parallel reduction using efficient iteration and synchronization
        for (int stride = num_q2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            int idx = threadIdx.x;
            while (idx < stride) {
                int offset = idx * num_atom2;
                f_ptxc[ii + offset] += f_ptxc[ii + offset + stride * num_atom2];
                f_ptyc[ii + offset] += f_ptyc[ii + offset + stride * num_atom2];
                f_ptzc[ii + offset] += f_ptzc[ii + offset + stride * num_atom2];
                idx += blockDim.x;
            }
        }
        __syncthreads();

        // Accumulate results back to global memory
        if (threadIdx.x == 0 && Ele[ii]) {
            Force[ii * 3    ] = f_ptxc[ii] * force_ramp;
            Force[ii * 3 + 1] = f_ptyc[ii] * force_ramp;
            Force[ii * 3 + 2] = f_ptzc[ii] * force_ramp;
        }
        __syncthreads();
    }
}