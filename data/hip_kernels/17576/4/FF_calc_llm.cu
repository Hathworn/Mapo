#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FF_calc (float *q_S_ref_dS, float *WK, float *vdW, int num_q, int num_ele, float c1, float r_m, float *FF_table, float rho) {

    __shared__ float q_pt, q_WK, C1, expC1;
    __shared__ float FF_pt[7]; // num_ele + 1, the last one for water.
    __shared__ float vdW_s[7];
    __shared__ float WK_s[66];
    __shared__ float C1_PI_43_rho;

    if (blockIdx.x >= num_q) return; // out of q range

    // Load WK into shared memory more efficiently
    for (int jj = threadIdx.x; jj < 11 * num_ele; jj += blockDim.x) {
        WK_s[jj] = WK[jj];
    }
    __syncthreads();

    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {
        q_pt = q_S_ref_dS[ii];
        q_WK = q_pt / (4.0 * PI);
        
        // Calculate FoXS C1 term 
        expC1 = -powf((4.0 * PI / 3.0), 1.5) * q_WK * q_WK * r_m * r_m * (c1 * c1 - 1.0) / (4.0 * PI);
        C1 = powf(c1, 3) * exp(expC1);
        C1_PI_43_rho = C1 * PI * 4.0 / 3.0 * rho;

        // Preload vdW data into shared memory
        for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
            vdW_s[jj] = vdW[jj];
        }
        __syncthreads();

        // Calculate Form factor for this block (or q vector)
        for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
            if (jj == num_ele) {
                // Water calculation
                FF_pt[jj] = WK_s[3 * 11 + 5];
                FF_pt[jj] += 2.0 * WK_s[5];
                float vdW_jj = vdW_s[jj];
                float exp_factor = exp(-PI * vdW_jj * vdW_jj * q_WK * q_WK);
                FF_pt[jj] -= C1_PI_43_rho * powf(vdW_jj, 3.0) * exp_factor;

#pragma unroll 5
                for (int kk = 0; kk < 5; kk++) {
                    float exp_term = exp(-WK_s[3 * 11 + kk + 6] * q_WK * q_WK);
                    FF_pt[jj] += WK_s[3 * 11 + kk] * exp_term;
                    FF_pt[jj] += WK_s[kk] * exp_term;
                    FF_pt[jj] += WK_s[kk] * exp_term;
                }
            } 
            else {
                // Element calculation
                FF_pt[jj] = WK_s[jj * 11 + 5];
                float vdW_jj = vdW_s[jj];
                float exp_factor = exp(-PI * vdW_jj * vdW_jj * q_WK * q_WK);
                FF_pt[jj] -= C1_PI_43_rho * powf(vdW_jj, 3.0) * exp_factor;

#pragma unroll 5
                for (int kk = 0; kk < 5; kk++) {
                    float exp_term = exp(-WK_s[jj * 11 + kk + 6] * q_WK * q_WK);
                    FF_pt[jj] += WK_s[jj * 11 + kk] * exp_term;
                }
            }
            FF_table[ii * (num_ele + 1) + jj] = FF_pt[jj];
        }
    }
}