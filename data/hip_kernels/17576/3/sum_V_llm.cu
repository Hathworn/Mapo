#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_V ( float *V, float *V_s, int num_atom, int num_atom2, int *Ele, float *vdW) {

    // Use a single loop to handle both operations and avoid multiple __syncthreads()
    int ii = blockIdx.x * blockDim.x + threadIdx.x;
    if (ii < num_atom2) {
        if (ii < num_atom) {
            int atomi = Ele[ii];
            if (atomi > 5) atomi = 0;
            V_s[ii] = V[ii] * 4.0 * PI * vdW[atomi] * vdW[atomi];
        } else {
            V_s[ii] = 0.0;
        }
    }

    // Improved reduction using a binary tree pattern
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        __syncthreads();
        if (threadIdx.x < stride && (threadIdx.x + stride) < num_atom2) {
            V_s[threadIdx.x] += V_s[threadIdx.x + stride];
        }
    }
    __syncthreads();

    // Output result from the first thread
    if (threadIdx.x == 0 && blockIdx.x == 0) printf("Convex contact area = %.3f A^2.\n", V_s[0]);
}