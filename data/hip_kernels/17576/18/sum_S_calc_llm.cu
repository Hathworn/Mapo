#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_S_calc (float *S_calcc, float *f_ptxc, float *f_ptyc, float *f_ptzc, float *S_calc, float *Aq, float *q_S_ref_dS, int num_q, int num_atom, int num_atom2, float alpha, float k_chi, float *sigma2) {
    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {
        // Optimized reduction using parallel reduction
        for (int stride = num_atom2 / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            for (int iAccum = threadIdx.x; iAccum < stride; iAccum += blockDim.x) {
                S_calcc[ii * num_atom2 + iAccum] += S_calcc[ii * num_atom2 + stride + iAccum];
            }
        }
        __syncthreads();

        S_calc[ii] = S_calcc[ii * num_atom2];

        // Simplified computation for Aq with shared memory usage
        __shared__ float shared_Aq;
        if (threadIdx.x == 0) {
            shared_Aq = S_calc[ii] - q_S_ref_dS[ii + num_q];
            shared_Aq *= -alpha;
            shared_Aq += q_S_ref_dS[ii + 2 * num_q];
            shared_Aq *= k_chi / sigma2[ii];
            shared_Aq += shared_Aq;
            Aq[ii] = shared_Aq;
        }
        __syncthreads();

        float local_Aq = shared_Aq;
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            f_ptxc[ii * num_atom2 + jj] *= local_Aq * alpha;
            f_ptyc[ii * num_atom2 + jj] *= local_Aq * alpha;
            f_ptzc[ii * num_atom2 + jj] *= local_Aq * alpha;
        }
    }
}