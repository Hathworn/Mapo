#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void create_FF_full_FoXS_surf_grad(float *FF_table, float *V, float c2, int *Ele, float *FF_full, float *surf_grad, int num_q, int num_ele, int num_atom, int num_atom2) {
    __shared__ float FF_pt[7];
    float hydration;

    for (int ii = blockIdx.x; ii < num_q + 1; ii += gridDim.x) {
        // Get form factor for this block (or q vector)
        if (ii < num_q) {
            // Efficient use of shared memory
            for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
                FF_pt[jj] = FF_table[ii * (num_ele + 1) + jj];
            }
        }
        __syncthreads();

        // Calculate hydration factor
        hydration = c2 * FF_pt[num_ele];
        __syncthreads();

        if (ii == num_q) {
            // Calculate surface gradient
            for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
                surf_grad[3 * jj] *= c2;
                surf_grad[3 * jj + 1] *= c2;
                surf_grad[3 * jj + 2] *= c2;
            }
        } else {
            // Calculate atomic form factor
            for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
                int atomt = Ele[jj];
                float form_factor = (atomt > 5) ? FF_pt[0] : FF_pt[atomt];

                // Optimize memory access
                FF_full[ii * num_atom2 + jj] = form_factor + hydration * V[jj];
            }
        }

        if (threadIdx.x == 0) {
            FF_full[ii * num_atom2 + num_atom + 1] = FF_pt[num_ele];
        }
    }
}