#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void create_FF_full_FoXS ( float *FF_table, float *V, float c2, int *Ele, float *FF_full, int num_q, int num_ele, int num_atom, int num_atom2) {
    __shared__ float FF_pt[7];  // Shared memory to store form factors
    float hydration;
    int ii = blockIdx.x;  // Use a local variable for block index

    if (ii < num_q) {
        // Load form factors into shared memory
        for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
            FF_pt[jj] = FF_table[ii * (num_ele + 1) + jj];
        }
        __syncthreads();  // Synchronize to ensure all threads have updated shared memory

        hydration = c2 * FF_pt[num_ele];  // Compute hydration once per block

        // Calculate atomic form factor for each atom
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            int atomt = Ele[jj];
            FF_full[ii * num_atom2 + jj] = FF_pt[(atomt > 5) ? 0 : atomt] + hydration * V[jj];
        }
    }
}