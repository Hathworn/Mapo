#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void create_FF_full_HyPred ( float *FF_table, float *V, float c2_F, float *c2_H, int *Ele, float *FF_full, int num_q, int num_ele, int num_atom, int num_atom2) {

    __shared__ float FF_pt[7];
    __shared__ float hydration[10];

    for (int ii = blockIdx.x; ii < num_q; ii += gridDim.x) {
        // Load form factors into shared memory for this q vector
        if (ii < num_q) {
            for (int jj = threadIdx.x; jj < num_ele + 1; jj += blockDim.x) {
                FF_pt[jj] = FF_table[ii*(num_ele+1)+jj];
            }
        }
        __syncthreads();

        // Precompute hydration values in shared memory
        for (int jj = threadIdx.x; jj < 10; jj += blockDim.x) {
            hydration[jj] = c2_F * c2_H[jj] * FF_pt[num_ele];
        }
        __syncthreads();

        // Compute form factors for each atom
        for (int jj = threadIdx.x; jj < num_atom; jj += blockDim.x) {
            int atomt = Ele[jj];
            float ff_value = FF_pt[atomt > 5 ? 0 : atomt]; // Use branchless access
            FF_full[ii*num_atom2 + jj] = ff_value;
            FF_full[ii*num_atom2 + jj] += hydration[atomt] * V[jj];
        }
    }
}