#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Use threadIdx.x for unique state assignment per thread
    int id = blockIdx.x * blockDim.x + threadIdx.x; 
    hiprand_init(seed, id, 0, &state[id]);  // Initialize state with unique id
}