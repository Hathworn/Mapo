#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function
__global__ void NormalizeOutput(const int num_elements, const int* __restrict__ original, int64_t* __restrict__ to_normalize, int64_t batch_index, int64_t class_index) {
    // Using shared memory to reduce global memory accesses
    extern __shared__ int64_t shared_memory[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        shared_memory[threadIdx.x * 3] = batch_index;
        shared_memory[threadIdx.x * 3 + 1] = class_index;
        shared_memory[threadIdx.x * 3 + 2] = static_cast<int64_t>(original[idx]);
        
        // Synchronize to ensure all threads have written their data to shared memory
        __syncthreads();

        // Write data from shared memory back to global memory
        to_normalize[idx * 3] = shared_memory[threadIdx.x * 3];
        to_normalize[idx * 3 + 1] = shared_memory[threadIdx.x * 3 + 1];
        to_normalize[idx * 3 + 2] = shared_memory[threadIdx.x * 3 + 2];
    }
}