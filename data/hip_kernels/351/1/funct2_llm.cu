#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/**
#Copyright 2013 Athanassios Kintsakis

#Licensed under the Apache License, Version 2.0 (the "License");
#you may not use this file except in compliance with the License.
#You may obtain a copy of the License at

#http://www.apache.org/licenses/LICENSE-2.0

#Unless required by applicable law or agreed to in writing, software
#distributed under the License is distributed on an "AS IS" BASIS,
#WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
#See the License for the specific language governing permissions and
#limitations under the License.

# Author: Athanassios Kintsakis
# contact: akintsakis@issel.ee.auth.gr, athanassios.kintsakis@gmail.com
**/
#define inf 9999

__global__ void funct2(int n, int k, float* x, int* qx) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if (ix < n * n) {
        int j = ix & (n - 1);
        
        // Load values into shared memory to minimize global memory access
        __shared__ float shared_x[1024];  // Adjust size as per block dimensions
        shared_x[threadIdx.x] = x[ix];
        __syncthreads();
        
        float temp2 = shared_x[threadIdx.x - j + k] + shared_x[k * n + j];
        
        // Atomic operations to prevent race conditions
        if (temp2 < shared_x[threadIdx.x]) {
            atomicMin(&x[ix], temp2);
            atomicExch(&qx[ix], k);
        }
    }
}