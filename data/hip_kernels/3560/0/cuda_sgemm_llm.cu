#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_sgemm(float* matrix_a, float* matrix_b, float* matrix_c, size_t M, size_t K, size_t N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize memory access pattern by using shared memory
    __shared__ float shared_a[BLOCK_SIZE_Y][BLOCK_SIZE_X];
    __shared__ float shared_b[BLOCK_SIZE_X][BLOCK_SIZE_Y];

    float sum = 0;
    for (int tile = 0; tile < (K + BLOCK_SIZE_X - 1) / BLOCK_SIZE_X; ++tile) {
        int tiledRow = row;
        int tiledCol = tile * BLOCK_SIZE_X + threadIdx.x;
        if (tiledRow < M && tiledCol < K) {
            shared_a[threadIdx.y][threadIdx.x] = matrix_a[INDEX(tiledRow, tiledCol, M, K)];
        } else {
            shared_a[threadIdx.y][threadIdx.x] = 0;
        }

        tiledRow = tile * BLOCK_SIZE_Y + threadIdx.y;
        tiledCol = col;
        if (tiledRow < K && tiledCol < N) {
            shared_b[threadIdx.y][threadIdx.x] = matrix_b[INDEX(tiledRow, tiledCol, K, N)];
        } else {
            shared_b[threadIdx.y][threadIdx.x] = 0;
        }
        
        __syncthreads();

        // Perform computation over tile
        for (int k = 0; k < BLOCK_SIZE_X; ++k) {
            sum += shared_a[threadIdx.y][k] * shared_b[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (col < N && row < M) {
        matrix_c[INDEX(row, col, M, N)] = sum;
    }
}