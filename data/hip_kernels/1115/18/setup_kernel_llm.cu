#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState *states, unsigned long seed) {
    // Calculate global thread id efficiently
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize the RNG state for each thread
    hiprand_init(seed + tid, tid, 0, &states[tid]);
}