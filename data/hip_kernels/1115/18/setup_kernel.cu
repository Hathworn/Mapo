#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_kernel ( hiprandState * states, unsigned long seed ){
const int tid = threadIdx.x + blockIdx.x * blockDim.x;
hiprand_init ( seed+tid*4, tid, 0, &states[tid] );
}