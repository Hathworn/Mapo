#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define R 4176
#define C 2048
#define N (R*C)
const int threadsPerBlock = 1024;
const int blocksPerGrid = 8352;

using namespace std;

hipError_t matrixOperation(double* c, const double* a, const double* b, unsigned int arrSize, int operation, float* kernel_runtime, float* GPU_transfer_time);
void CPUMatrixOperation(double* c, const double* a, const double* b, unsigned int arrSize, int operation);
long long start_timer();
long long stop_timer(long long start_time, const char *name);

// Optimize kernel to use coalesced memory access
__global__ void matrixMultiplyKernel(double* c, const double* a, const double* b) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < N; i += stride) {
        c[i] = a[i] * b[i];
    }
}