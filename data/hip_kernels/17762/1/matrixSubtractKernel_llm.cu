#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define R 4176
#define C 2048
#define N (R*C)
const int threadsPerBlock = 1024;
const int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

using namespace std;

hipError_t matrixOperation(double* c, const double* a, const double* b, unsigned int arrSize, int operation, float* kernel_runtime, float* GPU_transfer_time);
void CPUMatrixOperation(double* c, const double* a, const double* b, unsigned int arrSize, int operation);
long long start_timer();
long long stop_timer(long long start_time, const char *name);

// Optimized kernel to improve performance
__global__ void matrixSubtractKernel(double* c, const double* a, const double* b) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int total_threads = blockDim.x * gridDim.x;
    for (; tid < N; tid += total_threads) {
        c[tid] = a[tid] - b[tid];
    }
}