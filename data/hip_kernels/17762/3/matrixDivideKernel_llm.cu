#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// Author: Adriel Kim
// 6-27-2020
// Updated 7-5-2020

#define R 4176
#define C 2048
#define N (R*C)
const int threadsPerBlock = 1024;
const int blocksPerGrid = 8352;

using namespace std;

// Optimized kernel function for matrix division
__global__ void matrixDivideKernel(double* c, const double* a, const double* b) {
    // Use the tid calculated using grid-stride loop to ensure efficient usage of threads
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Eliminating the division operation within the while loop
    if (tid < N) {
        c[tid] = __ddiv_rd(a[tid], b[tid]); // Utilize fast device division
    }
    // Utilize a grid-stride loop to cover all elements efficiently
    tid += blockDim.x * gridDim.x;
    while (tid < N) {
        c[tid] = __ddiv_rd(a[tid], b[tid]); // Apply fast division operation
        tid += blockDim.x * gridDim.x;
    }
}