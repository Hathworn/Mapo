#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define R 4176
#define C 2048
#define N (R*C)
const int threadsPerBlock = 1024;
const int blocksPerGrid = 8352;

using namespace std;

hipError_t matrixOperation(double* c, const double* a, const double* b, unsigned int arrSize, int operation, float* kernel_runtime, float* GPU_transfer_time);
void CPUMatrixOperation(double* c, const double* a, const double* b, unsigned int arrSize, int operation);
long long start_timer();
long long stop_timer(long long start_time, const char *name);

__global__ void matrixAddKernel(double *c, const double *a, const double *b) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Stride for coalesced memory access
    for (int i = tid; i < N; i += stride) { // Unroll loop for efficiency
        c[i] = a[i] + b[i];
    }
}