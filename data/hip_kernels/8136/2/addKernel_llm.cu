#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global thread index to handle arrays larger than block size
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) { // Ensure we don't access out of bounds
        c[i] = a[i] + b[i];
    }
}