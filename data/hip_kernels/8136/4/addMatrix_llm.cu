#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addMatrix(int *c, int *a, int *b) {
    // Compute global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int index = idy * blockDim.x * gridDim.x + idx;

    // Perform matrix addition
    c[index] = a[index] + b[index];
}