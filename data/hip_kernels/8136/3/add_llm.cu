#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c) {
    // Use shared memory to store the results temporarily
    extern __shared__ int shared_mem[];
    int tid = threadIdx.x;
    
    // Single thread performs addition, optimized for future use of more threads
    if (tid == 0) {
        shared_mem[0] = a + b;
    }
    __syncthreads();
    
    // Copy result from shared memory to global memory
    if (tid == 0) {
        *c = shared_mem[0];
    }
}