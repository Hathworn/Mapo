#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_addcdiv(float *data, float value, float *src1, float *src2, long size)
{
    long k = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Check bounds and perform operation with minimal branching
    if (k < size)
    {
        float src2_val = src2[k];
        if(src2_val != 0) // Avoid division by zero
        {
            float src1_val = src1[k];
            data[k] += value * src1_val / src2_val;
        }
    }
}