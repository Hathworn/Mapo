#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_MSECriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, float norm, int nframe, int dim)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread is within bounds
    if (idx < nframe * dim) 
    {
        // Compute frame index and element index
        int k = idx / dim;
        int i = idx % dim;

        // Pointers to current frame
        float *gradInput_k = gradInput + k * dim;
        float *input_k = input + k * dim;
        float *target_k = target + k * dim;

        // Update gradInput
        gradInput_k[i] = norm * (input_k[i] - target_k[i]);
    }
}