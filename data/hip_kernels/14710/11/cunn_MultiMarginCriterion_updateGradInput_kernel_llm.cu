#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_MultiMarginCriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, int nframe, int dim, int sizeaverage)
{
    __shared__ float buffer[MULTIMARGIN_THREADS];
    int k = blockIdx.x;
    float *input_k = input + k*dim;
    float *gradInput_k = gradInput + k*dim;
    int target_k = ((int)target[k])-1;
    float input_target_k = input_k[target_k];
    float g = (sizeaverage ? 1.f/((float)dim) : 1.f);

    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;

    buffer[threadIdx.x] = 0;
    for (int i=i_start; i<i_end; i+=i_step)
    {
        float z = 1.f - input_target_k + input_k[i]; // Use 1.f for float arithmetic
        if(i == target_k)
            continue;

        if(z > 0.f) // Use 0.f for float arithmetic
        {
            buffer[threadIdx.x] -= g;
            gradInput_k[i] = g;
        }
        else
            gradInput_k[i] = 0.f; // Use 0.f for float arithmetic
    }

    __syncthreads();

    // Use a single atomic addition for reduction
    float gradInput_target_k = 0.f;
    if (threadIdx.x == 0) 
    {
        for (int i=0; i<blockDim.x; i++) 
            atomicAdd(&gradInput_target_k, buffer[i]);
            
        gradInput_k[target_k] = gradInput_target_k;
    }
}