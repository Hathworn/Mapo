#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Length of each data
__constant__ int gcT_size;
__constant__ int gcP_size;

// Threshold of the SW algorithm
__constant__ int gcThre;

// Data of the query
__constant__ char gcP_seq[1024];

// Cost and Gain
__constant__ int gcMatch;
__constant__ int gcMiss;
__constant__ int gcExtend;
__constant__ int gcBegin;

enum {
    Zero,
    Diagonal,
    Vertical,
    Horizon,
};

using namespace std;

__global__ void DP(char* dT_seq, char* dTrace, int* dScore) {
    // ThreadId = ptn point
    int id = threadIdx.x;
    // The acid in this thread
    char p = gcP_seq[id];
    // p-1 row line's value
    __shared__ int Hp_1[1024];
    __shared__ int Ep_1[1024];
    // Temporary
    int Hp_1_buf = 0;
    int Ep_1_buf = 0;
    // t-1 element value
    int Ht_1 = 0;
    int Ft_1 = 0;
    // p-1 t-1 element value
    int Ht_1p_1 = 0;
    // Initialize
    Hp_1[id] = 0;
    Ep_1[id] = 0;
    // Similar score
    int sim = 0;
    int point = id * gcT_size - id;

    // Calculate elements
    for (int t = -id; t < gcT_size; ++t) {
        // Get similar score
        if (t >= 0) {
            // Compare acids
            sim = (dT_seq[t] == p) ? gcMatch : gcMiss;

            // SW algorithm: Calculate each element
            int diagonal = Ht_1p_1 + sim;
            int horizonStart = Ht_1 + gcBegin;
            Ft_1 = max(horizonStart, Ft_1 + gcExtend);
            int verticalStart = Hp_1[id] + gcBegin;
            Ep_1_buf = max(verticalStart, Ep_1[id] + gcExtend);

            // Determine the max score and corresponding trace
            if (diagonal > Ep_1_buf && diagonal > Ft_1) {
                Ht_1 = diagonal;
                dTrace[point] = Diagonal;
            } else if (Ep_1_buf > Ft_1) {
                Ht_1 = Ep_1_buf;
                dTrace[point] = Vertical;
            } else {
                Ht_1 = Ft_1;
                dTrace[point] = Horizon;
            }

            // Zero is the max
            if (Ht_1 <= 0) {
                Ht_1 = 0;
                Ft_1 = 0;
                Ep_1_buf = 0;
                dTrace[point] = Zero;
            }

            // Hp-1 is next Ht-1p-1
            Ht_1p_1 = Hp_1[id];
            __syncthreads();

            // Set value needed for next calculation
            if (t >= 0) {
                Hp_1[id + 1] = Ht_1;
                Ep_1[id + 1] = Ep_1_buf;
            }

            // Update score if it exceeds the threshold
            if (Ht_1 >= gcThre && Ht_1 >= (dScore[t] & 0x0000ffff)) {
                dScore[t] = Ht_1 + (id << 16);
            }
        }
        ++point;
        __syncthreads();
    }
}