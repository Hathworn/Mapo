#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"



__global__ void rnd_init(size_t _count, hiprandState * _state)
{
int index = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y + (blockIdx.x * blockDim.x * blockDim.y * blockDim.z);
if (index >= _count)
{
return;
}
//Each thread gets same seed, a different sequence number, no offset
hiprand_init(1984, index, 0, &_state[index]);
}