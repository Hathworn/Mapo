#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel function
__global__ void rnd_init(size_t _count, hiprandState * _state)
{
    // Calculate linear index within the grid
    int index = blockIdx.x * blockDim.x + threadIdx.x 
                + (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x 
                + (blockIdx.z * blockDim.z + threadIdx.z) * gridDim.x * blockDim.x * gridDim.y * blockDim.y;
    
    // Guard against out-of-bounds threads
    if (index >= _count)
    {
        return;
    }
    
    // Initialize random state
    hiprand_init(1984, index, 0, &_state[index]);
}