#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory for reduction and reduce global memory accesses
__global__ void kernelGPU(float *R, float *G, float *B, float *Rin, float *Gin, float *Bin, int M, int N, int L) {

    int tId = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = M * N;
    
    // Declare shared memory
    __shared__ float sR[256];
    __shared__ float sG[256];
    __shared__ float sB[256];
    
    float rSum = 0.0f, gSum = 0.0f, bSum = 0.0f;
    int i;

    if (tId < M * N) {
        for (i = threadIdx.x; i < L; i += blockDim.x) {
            rSum += Rin[tId + i * stride];
            gSum += Gin[tId + i * stride];
            bSum += Bin[tId + i * stride];
        }
        
        // Reduction within the block
        sR[threadIdx.x] = rSum;
        sG[threadIdx.x] = gSum;
        sB[threadIdx.x] = bSum;
        
        __syncthreads();
                
        // Reduce sum within a block
        for (unsigned int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
            if (threadIdx.x < offset) {
                sR[threadIdx.x] += sR[threadIdx.x + offset];
                sG[threadIdx.x] += sG[threadIdx.x + offset];
                sB[threadIdx.x] += sB[threadIdx.x + offset];
            }
            __syncthreads();
        }
        
        // First thread in the block writes the result back to global memory
        if (threadIdx.x == 0) {
            R[blockIdx.x] = sR[0] / L;
            G[blockIdx.x] = sG[0] / L;
            B[blockIdx.x] = sB[0] / L;
        }
    }
}