#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_rng(hiprandState *states, int seed)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if tid is within the bounds before initializing
    if (tid < gridDim.x * blockDim.x) {
        hiprand_init(seed, tid, 0, &states[tid]);
    }
}