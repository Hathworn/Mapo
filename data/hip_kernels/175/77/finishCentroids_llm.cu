#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void finishCentroids(int* centroidMass, unsigned int* centroidCount, float* centroids) {
    int centroidNumber = blockIdx.y * blockDim.y + threadIdx.y;
    int dimensionNumber = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize checks by using consistent thread/block calculations
    if (centroidNumber < 64 && dimensionNumber < 34) {
        int idx = dimensionNumber * 64 + centroidNumber; // Precompute index
        float totalCount = static_cast<float>(centroidCount[centroidNumber]); // Use static_cast for clarity
        float mass = static_cast<float>(centroidMass[idx]); // Reuse precomputed index
        centroids[idx] = mass / (static_cast<float>(INTCONFACTOR) * totalCount);
    }
}