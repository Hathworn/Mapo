#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void integrateBins(int width, int height, int nbins, int* devImage, int binPitch, int* devIntegrals) {
    __shared__ int pixels[16];
    const int blockX = blockDim.y * blockIdx.x;
    const int threadX = threadIdx.y;
    const int bin = threadIdx.x;
    const int x = blockX + threadX;
    
    // Boundary check for valid x and bin
    if (x >= width || bin >= nbins) return;
    
    int accumulant = 0;
    int* imagePointer = devImage + x;
    int* outputPointer = devIntegrals + binPitch * x + bin;
    
    for (int y = 0; y < height; y++) {
        // Load pixels once per row into shared memory
        if (bin == 0) {
            pixels[threadX] = *imagePointer;
        }
        __syncthreads();
        
        // Using shared memory for caching pixel values
        if (pixels[threadX] == bin) accumulant++;
        
        *outputPointer = accumulant;
        imagePointer += width;
        outputPointer += width * binPitch;
    }
}