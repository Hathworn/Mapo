#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findCentroidsAtomicFreeReduce(int afLocal, int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount)
{
    int const af_id = blockIdx.x;
    int const cluster_id = blockIdx.y;
    int const filter_id = threadIdx.x;

    // Use shared memory to help reduce global memory access
    __shared__ int shared_mass[64];
    __shared__ int shared_count[64];
    
    // Initialize shared memory for this thread
    if (filter_id < 64) {
        shared_mass[filter_id] = 0;
        shared_count[filter_id] = 0;
    }
    __syncthreads();

    if (af_id == 0)
    {
        int idx0 = filter_id * 64 + cluster_id;

        for (int i = 0; i < gridDim.x; i++)
        {
            int idxother = i * gridDim.y * blockDim.x + idx0;

            // Accumulate results into shared memory
            atomicAdd(&shared_mass[filter_id], centroidMass[idxother]);
            atomicAdd(&shared_count[filter_id], centroidCount[idxother]);
        }

        __syncthreads(); // Ensure all threads have completed accumulation

        // Only one thread writes the result back to global memory
        if (filter_id < 64) {
            centroidMass[idx0] = shared_mass[filter_id];
            centroidCount[idx0] = shared_count[filter_id];
        }
    }
}