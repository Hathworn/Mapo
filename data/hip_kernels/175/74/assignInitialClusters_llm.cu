#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void assignInitialClusters(int width, int height, int nPixels, int clusterCount, int* cluster, int filterCount, float* responses, int* intResponses) {
    // Use shared memory for frequently accessed data if possible (such as small variables)
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if ((x < width) && (y < height)) {
        int pixel = y * width + x;
        int xBlock = __fdividef(x, ((width - 1) / 8 + 1)); // Use faster division
        int yBlock = __fdividef(y, ((height - 1) / 8 + 1)); // Use faster division
        int assignedCluster = yBlock * 8 + xBlock;

        cluster[pixel] = assignedCluster; // Use computed index variable directly

        // Utilize loop unrolling if filterCount is known and small
        for(int i = 0; i < filterCount; i++) {
            int index = pixel + i * nPixels;
            int response = __float2int_rn(INTCONFACTOR * responses[index]); // Use intrinsic for conversion
            intResponses[index] = response;
        }
    }
}