#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findCentroids(int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount) {
    __shared__ int localMasses[32*17];
    __shared__ unsigned int localCounts[32];

    int pixel = blockDim.x * blockIdx.x + threadIdx.x;

    // Initialize shared memory
    for (int i = threadIdx.x; i < 32; i += blockDim.x) {
        for (int j = 0; j < 17; j++) {
            localMasses[32 * j + i] = 0;
        }
        localCounts[i] = 0;
    }
    __syncthreads();

    if (pixel < nPixels) {
        int myCluster = cluster[pixel];
        int myIndex = pixel;

        // Accumulate mass for the first half
        for (int filter = 0; filter < 17; filter++) {
            int myElement = responses[myIndex];
            atomicAdd(localMasses + filter * 32 + myCluster, myElement);
            myIndex += nPixels;
        }
    }
    __syncthreads();

    // Update global centroid values for the first half
    for (int i = threadIdx.x; i < 32; i += blockDim.x) {
        for (int filter = 0; filter < 17; filter++) {
            atomicAdd(centroidMass + filter * 32 + i, localMasses[i + filter * 32]);
        }
    }
    __syncthreads();

    if (pixel < nPixels) {
        int myCluster = cluster[pixel];
        int myIndex = pixel + nPixels*17;

        // Accumulate mass for the second half
        for (int filter = 0; filter < 17; filter++) {
            int myElement = responses[myIndex];
            atomicAdd(localMasses + filter * 32 + myCluster, myElement);
            myIndex += nPixels;
        }
        atomicInc(localCounts + myCluster, 100000000);
    }
    __syncthreads();

    // Update global centroid values and counts for the second half
    for (int i = threadIdx.x; i < 32; i += blockDim.x) {
        for (int filter = 17; filter < 34; filter++) {
            atomicAdd(centroidMass + filter * 32 + i, localMasses[i + (filter - 17) * 32]);
        }
        atomicAdd(centroidCount + i, localCounts[i]);
    }
}