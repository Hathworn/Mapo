#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findCentroidsAtomicFreeLocal_64(int afLocal, int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount)
{
    int const af_id = blockIdx.x;
    int const cluster_id = blockIdx.y;
    int const filter_id = threadIdx.x;
    int* filter_responses = &responses[filter_id * nPixels];

    extern __shared__ int sharedMemory[]; // Shared memory for local updates
    int* shared_responses = sharedMemory;
    int* shared_count = &sharedMemory[blockDim.x];

    shared_responses[filter_id] = 0;
    shared_count[filter_id] = 0;

    __syncthreads(); // Synchronize before accessing shared memory

    int pixel_start = af_id * afLocal;
    int pixel_end = (af_id + 1) * afLocal;
    pixel_end = pixel_end > nPixels ? nPixels : pixel_end;

    for (int i = pixel_start; i < pixel_end; i++)
    {
        if (cluster[i] == cluster_id)
        {
            shared_responses[filter_id] += filter_responses[i];
            shared_count[filter_id]++;
        }
    }

    __syncthreads(); // Ensure all threads have updated shared memory

    int idx = af_id * gridDim.y * blockDim.x + filter_id * 64 + cluster_id;
    centroidMass[idx] = shared_responses[filter_id];
    centroidCount[idx] = shared_count[filter_id];
}