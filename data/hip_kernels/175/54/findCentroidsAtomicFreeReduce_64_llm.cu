#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findCentroidsAtomicFreeReduce_64(int afLocal, int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount)
{
    int const af_id = blockIdx.x;
    int const cluster_id = blockIdx.y;
    int const filter_id = threadIdx.x;

    int local_mass = 0;
    int local_count = 0;

    if (af_id == 0)
    {
        int idx0 = filter_id * 64 + cluster_id;

        // Use shared memory for faster local_mass and local_count aggregation
        __shared__ int shared_mass[64 * blockDim.y];
        __shared__ unsigned int shared_count[64 * blockDim.y];

        shared_mass[filter_id * blockDim.y + cluster_id] = 0;
        shared_count[filter_id * blockDim.y + cluster_id] = 0;
        
        for (int i = 0; i < gridDim.x; i++)
        {
            int idxother = i * gridDim.y * blockDim.x + idx0;
            atomicAdd(&shared_mass[filter_id * blockDim.y + cluster_id], centroidMass[idxother]);
            atomicAdd(&shared_count[filter_id * blockDim.y + cluster_id], centroidCount[idxother]);
        }

        // Only a single thread needs to write back the result
        if (filter_id == 0) {
            centroidMass[idx0] = shared_mass[cluster_id];
            centroidCount[idx0] = shared_count[cluster_id];
        }
    }
}