#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// vim: ts=4 syntax=cpp comments=


#define XBLOCK 16
#define YBLOCK 16

__global__ void mirrorImage_kernel(uint width, uint height, uint border, uint borderWidth, uint borderHeight, float* devInput, float* devOutput) {
    int x0 = blockDim.x * blockIdx.x + threadIdx.x;
    int y0 = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if within bounds
    if (x0 >= borderWidth || y0 >= borderHeight)
        return;

    // Calculate mirrored x1, y1 using conditional expressions
    int x1 = (x0 < border) ? (border - x0 - 1) : (x0 < border + width ? x0 - border : border + 2 * width - x0 - 1);
    int y1 = (y0 < border) ? (border - y0 - 1) : (y0 < border + height ? y0 - border : border + 2 * height - y0 - 1);

    // Write output
    devOutput[y0 * borderWidth + x0] = devInput[y1 * width + x1];
}