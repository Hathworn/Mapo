#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void finishCentroids(int* centroidMass, unsigned int* centroidCount, float* centroids) {
    int centroidNumber = blockIdx.y * blockDim.y + threadIdx.y;
    int dimensionNumber = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundaries once and store commonly used indices
    if (centroidNumber < 64 && dimensionNumber < 34) {
        int idx = dimensionNumber * 64 + centroidNumber;
        float totalCount = static_cast<float>(centroidCount[centroidNumber]);
        float mass = static_cast<float>(centroidMass[idx]);
        centroids[idx] = mass / (static_cast<float>(INTCONFACTOR) * totalCount);
    }
}