#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void combine_kernel(int nPixels, int cuePitchInFloats, float* devBg, float* devCga, float* devCgb, float* devTg, float* devMpb, float* devCombinedg) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int orientation = threadIdx.y;
    int orientedIndex = orientation * cuePitchInFloats + index;

    if (index < nPixels) {
        float accumulant = 0.0f;
        float accumulant2 = 0.0f;
        float* pointers[4] = { &devBg[orientedIndex], &devCga[orientedIndex], &devCgb[orientedIndex], &devTg[orientedIndex] };
        const int offsets[3] = { 0, 8 * cuePitchInFloats, 16 * cuePitchInFloats };

        #pragma unroll // Unroll the loop to enhance performance
        for (int i = 0; i < 3; ++i) {
            accumulant += pointers[0][offsets[i]] * coefficients[i] + 
                          pointers[1][offsets[i]] * coefficients[3 + i] + 
                          pointers[2][offsets[i]] * coefficients[6 + i] + 
                          pointers[3][offsets[i]] * coefficients[9 + i];
            accumulant2 += pointers[0][offsets[i]] * weights[i] + 
                           pointers[1][offsets[i]] * weights[3 + i] + 
                           pointers[2][offsets[i]] * weights[6 + i] + 
                           pointers[3][offsets[i]] * weights[9 + i];
        }
        
        devMpb[orientedIndex] = accumulant;
        devCombinedg[orientedIndex] = accumulant2;
    }
}