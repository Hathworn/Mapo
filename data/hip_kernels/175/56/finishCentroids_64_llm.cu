#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void finishCentroids_64(int* centroidMass, unsigned int* centroidCount, float* centroids) {
    int centroidNumber = blockIdx.y * blockDim.y + threadIdx.y;
    int dimensionNumber = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure threads are within valid range
    if (centroidNumber < 64 && dimensionNumber < 34) {
        // Use variables to reduce redundant calculations
        int index = dimensionNumber * 64 + centroidNumber;
        float mass = (float)centroidMass[index];
        float totalCount = (float)centroidCount[centroidNumber];
        
        // Calculate the centroid value
        centroids[index] = mass / ((float)INTCONFACTOR * totalCount);
    }
}