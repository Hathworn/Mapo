#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findCentroidsAtomicFreeReduce(int afLocal, int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount)
{
    int const af_id = blockIdx.x;
    int const cluster_id = blockIdx.y;
    int const filter_id = threadIdx.x;
    
    __shared__ int shared_mass[32]; // Use shared memory for reduction
    __shared__ int shared_count[32];

    shared_mass[filter_id] = 0;
    shared_count[filter_id] = 0;
    __syncthreads();
    
    if (af_id == 0)
    {
        int idx0 = filter_id * 32 + cluster_id;

        for (int i=0; i<gridDim.x; i++) {
            int idxother = i * gridDim.y * blockDim.x + idx0;

            atomicAdd(&shared_mass[filter_id], centroidMass[idxother]); // Use atomic operations on shared memory
            atomicAdd(&shared_count[filter_id], centroidCount[idxother]);
        }
        __syncthreads();

        if (filter_id == 0) { // Only one thread writes back to global memory
            for (int i = 1; i < blockDim.x; i++) {
                shared_mass[0] += shared_mass[i];
                shared_count[0] += shared_count[i];
            }
            centroidMass[idx0] = shared_mass[0];
            centroidCount[idx0] = shared_count[0];
        }
    }
}