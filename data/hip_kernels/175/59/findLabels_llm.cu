#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findLabels(int nPixels, int filterCount, int clusterCount, float* responses, float* centroids, int* clusters, int* changes) {
    __shared__ float sharedCentroids[34 * 32];
    __shared__ unsigned int localChanges;

    int x = blockDim.x * blockIdx.x + threadIdx.x;

    // Load centroids into shared memory for both set of labels (0-31 and 32-63)
    if (threadIdx.x < 64) {
        int base = (threadIdx.x < 32) ? 0 : 32;
        int idx = threadIdx.x % 32;
        for (int i = 0; i < 34; i++) {
            float element = centroids[i * 64 + idx + base];
            sharedCentroids[i * 32 + idx] = element;
        }
    }
    __syncthreads();

    float bestDistance = FLT_MAX;
    int bestLabel = -1;

    // Process pixels
    if (x < nPixels) {
        for (int segment = 0; segment < 2; segment++) {
            int offset = segment * 32;
            for (int label = 0; label < 32; label++) {
                float accumulant = 0.0f;
                int index = x;
                for (int dimension = 0; dimension < 34; dimension++) {
                    float diff = sharedCentroids[dimension * 32 + label + offset] - responses[index];
                    accumulant += diff * diff;
                    index += nPixels;
                }
                if (accumulant < bestDistance) {
                    bestDistance = accumulant;
                    bestLabel = label + offset;
                }
            }
            __syncthreads(); // Ensure all threads are in sync before next loading phase
        }

        int formerCluster = clusters[x];
        if (bestLabel != formerCluster) {
            atomicInc(&localChanges, 10000000);
        }
        clusters[x] = bestLabel;
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        atomicAdd(changes, localChanges);
    }
}