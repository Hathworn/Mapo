#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findCentroids(int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount) {
    __shared__ int localMasses[64*34]; // Allocate enough shared memory for all filters in one array
    __shared__ unsigned int localCounts[64];

    int pixel = blockDim.x * blockIdx.x + threadIdx.x;

    // Initialize localMasses and localCounts using a single loop
    for (int i = threadIdx.x; i < 64*34; i += blockDim.x) {
        localMasses[i] = 0;
    }
    if (threadIdx.x < 64) {
        localCounts[threadIdx.x] = 0;
    }
    __syncthreads();

    // Combine two separate loops over responses into one
    if (pixel < nPixels) {
        int myCluster = cluster[pixel];
        int myIndex = pixel;
        for (int filter = 0; filter < 34; filter++) {
            int myElement = responses[myIndex];
            atomicAdd(localMasses + filter * 64 + myCluster, myElement);
            myIndex += nPixels;
        }
        atomicInc(localCounts + myCluster, 100000000);
    }
    __syncthreads();

    // Combine two separate loops for updating centroidMass into one
    if (threadIdx.x < 64) {
        for (int filter = 0; filter < 34; filter++) {
            atomicAdd(centroidMass + filter * 64 + threadIdx.x, localMasses[threadIdx.x + filter * 64]);
        }
        atomicAdd(centroidCount + threadIdx.x, localCounts[threadIdx.x]);
    }
}