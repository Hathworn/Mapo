#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void assignInitialClusters(int width, int height, int nPixels, int clusterCount, int* cluster, int filterCount, float* responses, int* intResponses) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int pixel = y * width + x;

    // Ensure threads operate only on valid image coordinates
    if (x < width && y < height) {
        // Precompute block size values to simplify expressions
        int xBlock = x / ((width - 1) / 6 + 1);
        int yBlock = y / ((height - 1) / 6 + 1);
        
        int assignedCluster = yBlock * 6 + xBlock;

        // Clamp the assigned cluster to a maximum of 31
        assignedCluster = min(assignedCluster, 31);

        cluster[y * width + x] = assignedCluster;

        // Loop unrolling might improve performance, enable compile-time unrolling 
        #pragma unroll
        for (int i = 0; i < filterCount; i++) {
            int index = pixel + i * nPixels;
            int response = static_cast<int>(INTCONFACTOR * responses[index]);
            intResponses[index] = response;
        }
    }
}