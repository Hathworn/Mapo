#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void assignInitialClusters_64(int width, int height, int nPixels, int clusterCount, int* cluster, int filterCount, float* responses, int* intResponses) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int pixel = y * width + x;
    
    if ((x < width) && (y < height)) {
        // Precompute divisions to minimize repeated calculations
        int widthBlockSize = (width - 1) / 8 + 1;
        int heightBlockSize = (height - 1) / 8 + 1;
        
        int xBlock = x / widthBlockSize;
        int yBlock = y / heightBlockSize;
        int assignedCluster = yBlock * 8 + xBlock;
        
        cluster[pixel] = assignedCluster;
        
        int factor = INTCONFACTOR;  // Store constant in a register
        for(int i = 0; i < filterCount; i++) {
            int index = pixel + i * nPixels;
            int response = __float2int_rn(factor * responses[index]);  // Use intrinsic for floating-point to integer conversion
            intResponses[index] = response;
        }
    }
}