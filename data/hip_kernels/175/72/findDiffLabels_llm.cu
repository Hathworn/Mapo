#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findDiffLabels(float* devDiff, int diffPitchInFloats, int nPoints, int nClusters, int* devClusters, int* devChanges) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ unsigned int localChanges;
    
    // Initialize shared memory counter to 0
    if (threadIdx.x == 0) {
        localChanges = 0;
    }
    __syncthreads();

    if (x < nPoints) {
        int index = x;
        float minDistance = 1e20f; // Use a more precise representation for large numbers
        int minCluster = -1;
        
        // Iterate over clusters to find the closest one
        for (int cluster = 0; cluster < nClusters; cluster++) {
            float clusterDistance = devDiff[index];
            if (clusterDistance < minDistance) {
                minDistance = clusterDistance;
                minCluster = cluster;
            }
            index += diffPitchInFloats;
        }

        // Update the cluster assignment
        int previousCluster = devClusters[x];
        devClusters[x] = minCluster;
        
        // Atomically increment if there was a change in cluster assignment
        if (minCluster != previousCluster) {
            atomicAdd(&localChanges, 1);
        }
    }
    
    __syncthreads();
    
    // Accumulate changes from all threads in the block
    if (threadIdx.x == 0) {
        atomicAdd(devChanges, localChanges);
    }
}