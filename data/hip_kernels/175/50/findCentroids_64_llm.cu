#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findCentroids_64(int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount) {
    __shared__ int localMasses[64 * 17];
    __shared__ unsigned int localCounts[64];
    int pixel = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Use block-wide synchronization
    for (int i = threadIdx.x; i < 64 * 17; i += blockDim.x) {
        localMasses[i] = 0;
    }
    if (threadIdx.x < 64) {
        localCounts[threadIdx.x] = 0;
    }
    __syncthreads();
    
    // Process pixels and accumulate local masses
    if (pixel < nPixels) {
        int myCluster = cluster[pixel];
        int myIndex = pixel;
        for (int filter = 0; filter < 17; filter++) {
            int myElement = responses[myIndex];
            atomicAdd(localMasses + filter * 64 + myCluster, myElement);
            myIndex += nPixels;
        }
    }
    __syncthreads();
    
    // Aggregate to global centroid masses
    for (int i = threadIdx.x; i < 64 * 17; i += blockDim.x) {
        atomicAdd(centroidMass + i, localMasses[i]);
    }

    // Process for second set of filters
    __syncthreads();
    if (pixel < nPixels) {
        int myCluster = cluster[pixel];
        int myIndex = pixel + nPixels * 17;
        for (int filter = 0; filter < 17; filter++) {
            int myElement = responses[myIndex];
            atomicAdd(localMasses + filter * 64 + myCluster, myElement);
            myIndex += nPixels;
        }
        atomicInc(localCounts + myCluster, 0xFFFFFFFF);
    }
    __syncthreads();
    
    // Aggregate results for second set of filters
    for (int filter = threadIdx.x; filter < 34 * 64; filter += blockDim.x) {
        if (filter < 17 * 64) {
            atomicAdd(centroidMass + 64 * 17 + filter, localMasses[filter]);
        }
        if (filter < 64) {
            atomicAdd(centroidCount + filter, localCounts[filter]);
        }
    }
}