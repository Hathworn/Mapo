#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
// vim: ts=4 syntax=cpp comments=

#define XBLOCK 16
#define YBLOCK 16

__global__ void rgbUtoLab3F_kernel(int width, int height, float gamma, unsigned int* rgbU, float* devL, float* devA, float* devB) {
    int x0 = blockDim.x * blockIdx.x + threadIdx.x;
    int y0 = blockDim.y * blockIdx.y + threadIdx.y;

    if ((x0 < width) && (y0 < height)) {
        int index = y0 * width + x0;
        unsigned int rgb = rgbU[index];

        // Use bit manipulations directly and precompute division
        float divisor = 1.0f / 255.0f;
        float r = ((rgb & 0xff) * divisor);
        float g = (((rgb & 0xff00) >> 8) * divisor);
        float b = (((rgb & 0xff0000) >> 16) * divisor);

        // Inline powf for gamma correction to avoid multiple calls
        r = powf(r, gamma);
        g = powf(g, gamma);
        b = powf(b, gamma);

        float x = (0.412453f * r) + (0.357580f * g) + (0.180423f * b);
        float y = (0.212671f * r) + (0.715160f * g) + (0.072169f * b);
        float z = (0.019334f * r) + (0.119193f * g) + (0.950227f * b);

        // Pre-defined constants moved for optimization
        const float x_ref = 0.950456f;
        const float y_ref = 1.0f;
        const float z_ref = 1.088754f;
        const float threshold = 0.008856f;

        // Normalize using precomputed constants
        x /= x_ref;
        y /= y_ref;
        z /= z_ref;

        // Ternary operations simplified by inline ternary comparisons
        float fx = (x > threshold) ? cbrtf(x) : (7.787f * x + 0.13793f);
        float fy = (y > threshold) ? cbrtf(y) : (7.787f * y + 0.13793f);
        float fz = (z > threshold) ? cbrtf(z) : (7.787f * z + 0.13793f);

        // Compute Lab values; avoid repeated computations and improve readbility
        devL[index] = (y > threshold) ? (116.0f * cbrtf(y) - 16.0f) : (903.3f * y);
        devA[index] = 500.0f * (fx - fy);
        devB[index] = 200.0f * (fy - fz);
    }
}