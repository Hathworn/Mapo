#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findDiffLabelsAtomicFree(float* devDiff, int diffPitchInFloats, int nPoints, int nClusters, int* devClusters, int* devChanges) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if (x < nPoints) {
        int index = x;
        float minDistance = 10000000;
        int minCluster = -1;
        for(int cluster = 0; cluster < nClusters; cluster++) {
            float clusterDistance = devDiff[index];
            if (clusterDistance < minDistance) {
                minDistance = clusterDistance;
                minCluster = cluster;
            }
            index += diffPitchInFloats;
        }
        int previousCluster = devClusters[x];
        if (minCluster != previousCluster) {
            devClusters[x] = minCluster;
            
            // Atomic operation to avoid race condition
            atomicAdd(devChanges, 1);
        }
    }
}