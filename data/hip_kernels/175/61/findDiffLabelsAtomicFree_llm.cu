#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findDiffLabelsAtomicFree(float* devDiff, int diffPitchInFloats, int nPoints, int nClusters, int* devClusters, int* devChanges) {

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if (x < nPoints) {
        int index = x;
        float minDistance = 10000000;
        int minCluster = -1;

        // Optimize loop by using float* to traverse devDiff
        float* clusterDistancePtr = &devDiff[x];
        for(int cluster = 0; cluster < nClusters; cluster++) {
            float clusterDistance = *clusterDistancePtr;  // Directly access via pointer
            if (clusterDistance < minDistance) {
                minDistance = clusterDistance;
                minCluster = cluster;
            }
            clusterDistancePtr += diffPitchInFloats;  // Use pointer arithmetic for next cluster
        }

        int previousCluster = devClusters[x];
        devClusters[x] = minCluster;
        
        // Use atomicAdd to safely update devChanges without race conditions
        if (minCluster != previousCluster) {
            atomicAdd(devChanges, 1);
        }
    }
}