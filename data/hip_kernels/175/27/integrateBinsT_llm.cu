#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void integrateBinsT(int width, int height, int nbins, int binPitch, int* devIntegrals) {
    const int blockY = blockDim.y * blockIdx.x;
    const int threadY = threadIdx.y;
    const int bin = threadIdx.x;
    const int y = blockY + threadY;

    // Early return if out of bounds
    if (y >= height || bin >= nbins) return;

    int* imagePointer = devIntegrals + (y * width + bin) * binPitch;
    int accumulant = 0;

    // Loop unrolling for better performance
    for(int x = 0; x < width; x += 4) {
        if (x < width) accumulant += *imagePointer; *imagePointer = accumulant; imagePointer += binPitch;
        if (x + 1 < width) accumulant += *imagePointer; *imagePointer = accumulant; imagePointer += binPitch;
        if (x + 2 < width) accumulant += *imagePointer; *imagePointer = accumulant; imagePointer += binPitch;
        if (x + 3 < width) accumulant += *imagePointer; *imagePointer = accumulant; imagePointer += binPitch;
    }
}