#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fillImage(int width, int height, int value, int* devOutput) {
    // Calculate a single linear index for the pixel to improve cache usage
    int index = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x + gridDim.x * blockDim.y * threadIdx.y;
    
    // Check the boundary condition and assign value
    if (index < width * height) {
        devOutput[index] = value;
    }
}