#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// vim: ts=4 syntax=cpp comments=

#define XBLOCK 16
#define YBLOCK 16

__global__ void normalizeLab_kernel(uint width, uint height, float* devL, float* devA, float* devB) {
    int x0 = blockDim.x * blockIdx.x + threadIdx.x;
    int y0 = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if within bounds
    if (x0 < width && y0 < height) {
        int index = y0 * width + x0;
        const float ab_min = -73.0f;
        const float ab_max = 95.0f;
        const float ab_range = ab_max - ab_min;

        // Normalize and clamp Lab image values
        float l_val = fminf(fmaxf(devL[index] / 100.0f, 0.0f), 1.0f);
        float a_val = fminf(fmaxf((devA[index] - ab_min) / ab_range, 0.0f), 1.0f);
        float b_val = fminf(fmaxf((devB[index] - ab_min) / ab_range, 0.0f), 1.0f);

        devL[index] = l_val;
        devA[index] = a_val;
        devB[index] = b_val;
    }
}