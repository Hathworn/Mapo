#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findLabels(int nPixels, int filterCount, int clusterCount, float* responses, float* centroids, int* clusters, int* changes) {
    __shared__ float sharedCentroids[34 * 32];
    __shared__ unsigned int localChanges;

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Use single shared memory load
    if (threadIdx.x < 32) {
        for (int i = 0; i < 34; i++) {
            sharedCentroids[i * 32 + threadIdx.x] = centroids[i * 64 + threadIdx.x];
            sharedCentroids[i * 32 + threadIdx.x + 32] = centroids[i * 64 + threadIdx.x + 32];
        }
    }
    __syncthreads();

    int bestLabel = -1;
    float bestDistance = 1e10f;

    if (x < nPixels) {
        for (int label = 0; label < 64; label++) { // Loop through all 64 labels
            float accumulant = 0.0f;
            int index = x;
            for (int dimension = 0; dimension < 34; dimension++) {
                float diff = sharedCentroids[dimension * 32 + (label % 32)] - responses[index];
                accumulant += diff * diff;
                index += nPixels;
            }
            if (accumulant < bestDistance) {
                bestLabel = label;
                bestDistance = accumulant;
            }
        }
        int formerCluster = clusters[x];
        if (bestLabel != formerCluster) {
            atomicInc(&localChanges, 10000000);
        }
        clusters[x] = bestLabel;
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        atomicAdd(changes, localChanges);
    }
}