#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void combine_kernel(int nPixels, int cuePitchInFloats, float* devBg, float* devCga, float* devCgb, float* devTg, float* devMpb, float* devCombinedg) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int orientation = threadIdx.y;
    int orientedIndex = orientation * cuePitchInFloats + index;

    if (index < nPixels) {
        float accumulant = 0.0;
        float accumulant2 = 0.0;

        // Use a loop for repeated operations to improve code maintainability
        float* sources[] = {devBg, devCga, devCgb, devTg};
        int numSources = sizeof(sources) / sizeof(sources[0]);

        #pragma unroll // Unroll loop for performance improvement
        for (int i = 0; i < numSources; ++i) {
            float* pointer = &sources[i][orientedIndex];
            int baseIndex = i * 3;
            
            #pragma unroll // Unroll inner loop for performance
            for (int j = 0; j < 3; ++j) {
                accumulant += *pointer * coefficients[baseIndex + j];
                accumulant2 += *pointer * weights[baseIndex + j];
                pointer += 8 * cuePitchInFloats; // Advance pointer as before
            }
        }

        devMpb[orientedIndex] = accumulant;
        devCombinedg[orientedIndex] = accumulant2;
    }
}