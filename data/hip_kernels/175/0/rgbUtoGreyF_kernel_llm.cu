#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define XBLOCK 16
#define YBLOCK 16

__global__ void rgbUtoGreyF_kernel(int width, int height, unsigned int* rgbU, float* grey) {
    // Efficiently calculate global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int index = y * width + x;
        unsigned int rgb = rgbU[index];

        // Use constant values for RGB to greyscale conversion
        const float r_weight = 0.29894;
        const float g_weight = 0.58704;
        const float b_weight = 0.11402;
        
        // Use bitwise operations and multiplication to enhance readability and potentially optimize execution
        float r = (rgb & 0xFF) * (1.0 / 255.0);
        float g = ((rgb >> 8) & 0xFF) * (1.0 / 255.0);
        float b = ((rgb >> 16) & 0xFF) * (1.0 / 255.0);
        
        // Calculate greyscale value using pre-defined weights
        grey[index] = r_weight * r + g_weight * g + b_weight * b;
    }
}