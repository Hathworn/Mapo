#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void assignInitialClusters(int width, int height, int nPixels, int clusterCount, int* cluster, int filterCount, float* responses, int* intResponses) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Check if thread is within bounds
    if ((x < width) && (y < height)) {
        int pixel = y * width + x;
        int xBlock = x / ((width - 1) / 8 + 1);
        int yBlock = y / ((height - 1) / 8 + 1);
        int assignedCluster = yBlock * 8 + xBlock;

        cluster[pixel] = assignedCluster;  // Use pixel index directly

        // Unroll the loop for better performance
        #pragma unroll
        for(int i = 0; i < filterCount; i++) {
            int index = pixel + i * nPixels;
            intResponses[index] = static_cast<int>(INTCONFACTOR * responses[index]);
        }
    }
}