#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findCentroidsAtomicFreeLocal(int afLocal, int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount)
{
    int const af_id = blockIdx.x;
    int const cluster_id = blockIdx.y;
    int const filter_id = threadIdx.x;
    extern __shared__ int shared_data[]; // Use shared memory for faster access
    int* shared_responses = &shared_data[filter_id * afLocal];

    int local_responses = 0;
    int local_count = 0;

    int pixel_start = af_id * afLocal;
    int pixel_end = min((af_id + 1) * afLocal, nPixels); // Use min for boundary check

    for (int i = pixel_start; i < pixel_end; i++)
    {
        if (cluster[i] == cluster_id)
        {
            local_responses += responses[filter_id*nPixels + i]; // Direct index calculation
            local_count++;
        }
    }

    shared_responses[filter_id] = local_responses; // Store intermediate results in shared memory
    __syncthreads(); // Ensure all threads have written to shared memory

    if (filter_id == 0) // Only one thread per block writes back to global memory
    {
        int idx = af_id * gridDim.y * blockDim.x + cluster_id;
        for (int i = 0; i < blockDim.x; i++) {
            centroidMass[idx] += shared_responses[i]; // Accumulate results from shared memory
            centroidCount[idx] += (i == filter_id ? local_count : 0); // Use the first thread to update the count
        }
    }
}