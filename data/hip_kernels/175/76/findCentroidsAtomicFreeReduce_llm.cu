#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findCentroidsAtomicFreeReduce(int afLocal, int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount)
{
    int const af_id = blockIdx.x;
    int const cluster_id = blockIdx.y;
    int const filter_id = threadIdx.x;

    // Using shared memory for local reductions
    __shared__ int shared_mass[64];
    __shared__ int shared_count[64];

    int local_mass = 0;
    int local_count = 0;

    if (af_id == 0)
    {
        int idx0 = filter_id * 64 + cluster_id;

        for (int i = filter_id; i < gridDim.x; i += blockDim.x)
        {
            int idxother = i * gridDim.y * blockDim.x + idx0;

            local_mass += centroidMass[idxother];
            local_count += centroidCount[idxother];
        }

        // Store local results in shared memory
        shared_mass[filter_id] = local_mass;
        shared_count[filter_id] = local_count;

        // Synchronize threads within block
        __syncthreads();

        // Reduction within block (assuming blockDim.x is 64, adjust as necessary)
        if (filter_id < 32)
        {
            for (int offset = 32; offset > 0; offset /= 2)
            {
                if (filter_id < offset)
                {
                    shared_mass[filter_id] += shared_mass[filter_id + offset];
                    shared_count[filter_id] += shared_count[filter_id + offset];
                }
                __syncthreads();
            }
        }

        // Write final results from shared memory back to global memory
        if (filter_id == 0)
        {
            centroidMass[idx0] = shared_mass[0];
            centroidCount[idx0] = shared_count[0];
        }
    }
}