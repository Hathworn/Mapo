#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMaxMinPerGrid(int p_nGridSize, int p_nEigNum, float* p_devMax, float* p_devMin, float* p_devReduceMax, float* p_devReduceMin, int p_nMaxLevel)
{
    // Declare shared memory for reduction
    extern __shared__ float sharedMem[];
    float* MaxReduce = sharedMem;
    float* MinReduce = &sharedMem[XBLOCK * (MAXEIGNUM - 1)];

    int taskPerTh = (p_nGridSize + XBLOCK - 1) / XBLOCK; // Calculate tasks per thread
    int tid = threadIdx.x;

    // Initial assignment in shared memory
    for (int i = 0; i < p_nEigNum - 1 && tid < p_nGridSize; i++) {
        MaxReduce[i * XBLOCK + tid] = p_devMax[tid + i * p_nGridSize];
        MinReduce[i * XBLOCK + tid] = p_devMin[tid + i * p_nGridSize];
    }

    // First reduction: iteratively check all taskPerThreads
    for (int i = 1; i < taskPerTh; i++) {
        int curIndex = tid + i * XBLOCK;
        if (curIndex < p_nGridSize) {
            for (int j = 0; j < p_nEigNum - 1; j++) {
                float curMax = p_devMax[curIndex + j * p_nGridSize];
                float curMin = p_devMin[curIndex + j * p_nGridSize];
                if (MaxReduce[j * XBLOCK + tid] < curMax) {
                    MaxReduce[j * XBLOCK + tid] = curMax;
                }
                if (MinReduce[j * XBLOCK + tid] > curMin) {
                    MinReduce[j * XBLOCK + tid] = curMin;
                }
            }
        }
    }
    __syncthreads();

    // Subsequent reductions: Use bit manipulation for efficient reduction
    for (int d = 64; d > 0; d >>= 1) {
        if (tid < d) {
            int offset = tid + d;
            if (offset < p_nGridSize) {
                for (int i = 0; i < p_nEigNum - 1; i++) {
                    if (MaxReduce[i * XBLOCK + tid] < MaxReduce[i * XBLOCK + offset]) {
                        MaxReduce[i * XBLOCK + tid] = MaxReduce[i * XBLOCK + offset];
                    }
                    if (MinReduce[i * XBLOCK + tid] > MinReduce[i * XBLOCK + offset]) {
                        MinReduce[i * XBLOCK + tid] = MinReduce[i * XBLOCK + offset];
                    }
                }
            }
        }
        __syncthreads();
    }

    // Store results back to global memory by the first thread
    if (tid == 0)
    {
        for (int i = 0; i < p_nEigNum - 1; i++)
        {
            p_devReduceMax[i] = MaxReduce[i * XBLOCK];
            p_devReduceMin[i] = MinReduce[i * XBLOCK];
        }
    }
}