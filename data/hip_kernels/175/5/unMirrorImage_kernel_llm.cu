#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// vim: ts=4 syntax=cpp comments=

#define XBLOCK 16
#define YBLOCK 16

// Optimized kernel function
__global__ void unMirrorImage_kernel(uint width, uint height, uint border, uint borderWidth, uint borderHeight, float* devInput, float* devOutput) {
    int x0 = blockIdx.x * blockDim.x + threadIdx.x;
    int y0 = blockIdx.y * blockDim.y + threadIdx.y;

    // Use shared memory for coalesced memory access (if necessary)
    // __shared__ float sharedData[...];

    if ((x0 < borderWidth) && (y0 < borderHeight)) {
        int x1 = x0 + border;
        int y1 = y0 + border;

        // Direct memory access with reduced arithmetic operations
        devOutput[y0 * borderWidth + x0] = devInput[y1 * width + x1];
    }
}