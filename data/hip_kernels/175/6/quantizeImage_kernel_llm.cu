#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// vim: ts=4 syntax=cpp comments=

#define XBLOCK 16
#define YBLOCK 16

__global__ void quantizeImage_kernel(uint width, uint height, uint nbins, float* devInput, int* devOutput) {
    // Calculate global thread index for x and y directions
    int x0 = blockIdx.x * blockDim.x + threadIdx.x;
    int y0 = blockIdx.y * blockDim.y + threadIdx.y;

    // Check for boundary conditions
    if (x0 >= width || y0 >= height) return;

    // Calculate the 1D index for the current thread
    int index = y0 * width + x0;

    // Fetch input once and compute the quantized output
    float input = devInput[index];
    int output = min((int)(input * (float)nbins), nbins - 1);

    // Store the result in the output array
    devOutput[index] = output;
}