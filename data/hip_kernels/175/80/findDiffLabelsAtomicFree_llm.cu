#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findDiffLabelsAtomicFree(float* devDiff, int diffPitchInFloats, int nPoints, int nClusters, int* devClusters, int* devChanges) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    if (x < nPoints) {
        int index = x;
        float minDistance = 1e10f;  // Initialize with a large float value
        int minCluster = -1;

        // Unrolling the loop, assuming nClusters is a small constant
        for (int cluster = 0; cluster < nClusters; ++cluster) {
            float clusterDistance = devDiff[index];
            if (clusterDistance < minDistance) {
                minDistance = clusterDistance;
                minCluster = cluster;
            }
            index += diffPitchInFloats;
        }

        int previousCluster = devClusters[x];
        devClusters[x] = minCluster;

        // Using atomic operation to update devChanges safely
        if (minCluster != previousCluster) {
            atomicAdd(devChanges, 1);
        }
    }
}