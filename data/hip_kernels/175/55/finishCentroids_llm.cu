#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void finishCentroids(int* centroidMass, unsigned int* centroidCount, float* centroids) {
    // Calculate 1D index instead of using two separate dimensions
    int idx = blockIdx.y * blockDim.y + threadIdx.y;
    int idy = blockIdx.x * blockDim.x + threadIdx.x;
    int flatIndex = idy * 32 + idx;
    
    // Improved boundary check
    if (idx < 32 && idy < 34) {
        float totalCount = static_cast<float>(centroidCount[idx]);
        float mass = static_cast<float>(centroidMass[flatIndex]);
        centroids[flatIndex] = mass / (static_cast<float>(INTCONFACTOR) * totalCount);
    }
}