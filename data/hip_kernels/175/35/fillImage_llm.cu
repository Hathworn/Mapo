#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function for filling an image with a value
__global__ void fillImage(int width, int height, int value, int* devOutput) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    // Calculate the global index and check bounds
    if (x < width && y < height) {
        int index = y * width + x; // Calculate linear index only if necessary
        devOutput[index] = value;
    }
}