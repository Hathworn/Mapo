#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findCentroidsAtomicFreeLocal(int afLocal, int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount)
{
    // Cache data in shared memory to minimize global accesses
    __shared__ int shared_responses[256]; // Adjust size based on actual use and occupancy
    __shared__ int shared_count[256];     // Adjust size based on block size and usage
    
    int const af_id = blockIdx.x;
    int const cluster_id = blockIdx.y;
    int const filter_id = threadIdx.x;
    int* filter_responses = &responses[filter_id * nPixels];
    
    int local_responses = 0;
    int local_count = 0;
    
    int pixel_start = af_id * afLocal;
    int pixel_end = (af_id + 1) * afLocal;
    
    pixel_end = pixel_end > nPixels ? nPixels : pixel_end;
    
    for (int i = pixel_start; i < pixel_end; i++)
    {
        if (cluster[i] == cluster_id)
        {
            local_responses += filter_responses[i];
            local_count++;
        }
    }
    
    // Use shared memory to accumulate results from threads
    shared_responses[filter_id] = local_responses;
    shared_count[filter_id] = local_count;
    
    __syncthreads();
    
    // Reduce phase for accumulation within block
    if (filter_id == 0)
    {
        int block_responses = 0;
        int block_count = 0;
        for (int i = 0; i < blockDim.x; i++)
        {
            block_responses += shared_responses[i];
            block_count += shared_count[i];
        }

        int idx = af_id * gridDim.y * blockDim.x + filter_id * 64 + cluster_id;
        // Write results from block to global memory
        centroidMass[idx] = block_responses;
        centroidCount[idx] = block_count;
    }
}