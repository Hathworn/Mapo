#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void assignColIds(int* colIds, const int* colOffsets) {
    int myId = blockIdx.x;
    int start = colOffsets[myId];
    int end = colOffsets[myId + 1];
    int totalThreads = blockDim.x * gridDim.x;
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;

    // Use globalId to distribute threads more evenly across blocks
    for (int id = start + globalId - myId * blockDim.x; id < end; id += totalThreads) {
        colIds[id] = myId;
    }
}