#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void permute_colors_kernel(int num_rows, int *row_colors, int *color_permutation)
{
    int row_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize using a stride loop and remove redundant calculations
    int stride = blockDim.x * gridDim.x;
    for ( ; row_id < num_rows ; row_id += stride )
    {
        int color = row_colors[row_id];
        // Use __ldg if supported for better cache efficiency
        #if __CUDA_ARCH__ >= 350
        color = __ldg(color_permutation + color);
        #else
        color = color_permutation[color];
        #endif
        row_colors[row_id] = color;
    }
}