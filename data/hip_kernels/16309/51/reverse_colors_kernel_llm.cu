#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reverse_colors_kernel(int num_rows, int max_color, int *row_colors)
{
    int row_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize loop with strided access pattern
    for (int i = row_id; i < num_rows; i += blockDim.x * gridDim.x)
    {
        int color = row_colors[i];

        // Reverse color only if it's greater than 0
        if (color > 0)
        {
            color = max_color - color + 1;
        }

        row_colors[i] = color;
    }
}