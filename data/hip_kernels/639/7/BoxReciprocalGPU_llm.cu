#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BoxReciprocalGPU(double *gpu_prefact, double *gpu_sumRnew, double *gpu_sumInew, double *gpu_energyRecip, int imageSize)
{
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    if(threadID >= imageSize) return;

    // Use register variables for improved memory access speed
    double sumR = gpu_sumRnew[threadID];
    double sumI = gpu_sumInew[threadID];
    double prefact = gpu_prefact[threadID];

    // Compute energy in registers before writing to global memory
    gpu_energyRecip[threadID] = (sumR * sumR + sumI * sumI) * prefact;
}