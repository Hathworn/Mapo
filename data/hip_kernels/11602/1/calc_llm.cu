#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calc(float *d_D, int n, int k) { // Kernel
    __shared__ float s_d[3 * 256]; // Shared memory for block

    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global indices
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int b_index = 3 * (threadIdx.x + blockDim.x * threadIdx.y); // Shared memory index

    // Load data into shared memory
    s_d[b_index] = d_D[i + j * n];
    s_d[b_index + 1] = d_D[i + k * n];
    s_d[b_index + 2] = d_D[k + j * n];
    __syncthreads(); // Ensure all threads have loaded their data

    // Compute new distance
    if (s_d[b_index] > s_d[b_index + 1] + s_d[b_index + 2])
        s_d[b_index] = s_d[b_index + 1] + s_d[b_index + 2];
    __syncthreads(); // Ensure computation completes before writing back 

    // Store result back to global memory
    d_D[i + j * n] = s_d[b_index];
}