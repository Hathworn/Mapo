#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef INFINITY
/* INFINITY is supported */
#endif

float **A, **D, *d2; //Table A distance, D minimum distance,d2 tempTable 1-d

__global__ void calc(float *d_D, int n, int k){
    // Calculate global thread indices for better hardware utilization
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure indices are within bounds to prevent out-of-bounds access
    if (i < n && j < n) {
        // Perform the core computation with reduced memory accesses
        float ik = d_D[i + k * n];
        float kj = d_D[k + j * n];
        float ij = d_D[i + j * n];
        
        if (ij > ik + kj) {
            d_D[i + j * n] = ik + kj;
        }
    }
}