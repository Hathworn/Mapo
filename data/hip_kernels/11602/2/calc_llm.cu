#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calc(float *d_D, int n, int k) {
    __shared__ float s_d[4*3*256];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int b_index = 4 * 3 * (threadIdx.x + blockDim.x * threadIdx.y);
    int istep = blockDim.x * gridDim.x, jstep = blockDim.y * gridDim.y;

    int l, m, v = 0;

    #pragma unroll
    for (l = 0; l < 2; l++)
    #pragma unroll
    for (m = 0; m < 2; m++) {
        int idx1 = (i + l * istep) + (j + m * jstep) * n;
        int idx2 = (i + l * istep) + k * n;
        int idx3 = k + (j + m * jstep) * n;

        s_d[b_index + 3 * v] = d_D[idx1];
        s_d[b_index + (3 * v + 1)] = d_D[idx2];
        s_d[b_index + (3 * v + 2)] = d_D[idx3];
        v++;
    }

    #pragma unroll
    for (v = 0; v < 4; v++) {
        float a = s_d[b_index + 3 * v];
        float b = s_d[b_index + (3 * v + 1)];
        float c = s_d[b_index + (3 * v + 2)];
        if (a > b + c) s_d[b_index + 3 * v] = b + c;
    }

    v = 0;
    #pragma unroll
    for (l = 0; l < 2; l++)
    #pragma unroll
    for (m = 0; m < 2; m++) {
        d_D[(i + l * istep) + (j + m * jstep) * n] = s_d[b_index + 3 * v];
        v++;
    }
}