#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void createLookupKernel(const int* inds, int total, int* output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Utilize shared memory to reduce global memory accesses
    __shared__ int sharedInds[256]; // Ensure this matches blockDim.x
    
    if (idx < total) {
        sharedInds[threadIdx.x] = inds[idx];
        __syncthreads(); // Ensure all threads have written their inds
        
        output[sharedInds[threadIdx.x]] = idx;
    }
}