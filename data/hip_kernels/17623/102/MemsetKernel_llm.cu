#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate the unique global thread index for the 2D grid
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check bounds in a single comparison
    if (i < h && j < w)
    {
        int pos = i * w + j;
        image[pos] = value;
    }
}