#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Use complete if-condition to reduce branch divergence
    if (i < h && j < w) {
        const int pos = i * s + j;
        float scale = normalization_factor[pos];
        float invScale = (scale != 0.0f) ? (1.0f / scale) : 1.0f;
        image[pos] *= invScale;
    }
}