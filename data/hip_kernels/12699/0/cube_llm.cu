#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA runtime

__global__ void cube(float *d_out, float *d_in) {
    // Use blockIdx.x to handle larger arrays beyond one block
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Cube computation
    float f = d_in[idx];
    d_out[idx] = f * f * f;
}