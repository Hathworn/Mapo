#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Limited version of checkCudaErrors from hip/hip_runtime_api.h

#define checkCudaErrors(val) check_errors((val), #val, __FILE__, __LINE__)

__global__ void render_init(int width, int length, hiprandState *rand_state) {
    // Compute the global index using single calculation
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Early exit for threads outside of target range
    if (i >= width || j >= length) return;

    int index = j * width + i;
    // Initialize the random state
    hiprand_init(1984, index, 0, &rand_state[index]);
}