#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_matrix_mult(int *a, int *b, int *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;

    // Check bounds before computation
    if (col < k && row < m)
    {
        // Unroll loop for potential performance improvement
        int i = 0;
        for (; i <= n - 4; i += 4)
        {
            sum += a[row * n + i] * b[i * k + col];
            sum += a[row * n + i + 1] * b[(i + 1) * k + col];
            sum += a[row * n + i + 2] * b[(i + 2) * k + col];
            sum += a[row * n + i + 3] * b[(i + 3) * k + col];
        }
        // Handle the remaining elements
        for (; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}