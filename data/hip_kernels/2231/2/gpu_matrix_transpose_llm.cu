#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_matrix_transpose(int* mat_in, int* mat_out, unsigned int rows, unsigned int cols)
{
    // Calculate the global index for each thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Combine condition checks using logical operators for efficiency
    if (idx < cols && idy < rows)
    {
        // Reduce shared computation of positions
        unsigned int pos = idy * cols + idx;
        unsigned int trans_pos = idx * rows + idy;

        // Directly assign transposed value
        mat_out[trans_pos] = mat_in[pos];
    }
}