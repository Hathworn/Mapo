#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_square_matrix_mult(int *d_a, int *d_b, int *d_result, int n)
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;

    for (int sub = 0; sub < (n + BLOCK_SIZE - 1) / BLOCK_SIZE; ++sub) // Ensure full grid coverage
    {
        int idx_a = row * n + sub * BLOCK_SIZE + threadIdx.x;
        int idx_b = (sub * BLOCK_SIZE + threadIdx.y) * n + col;

        // Use ternary operator for concise bounds check and assignment
        tile_a[threadIdx.y][threadIdx.x] = (idx_a < n * n) ? d_a[idx_a] : 0;
        tile_b[threadIdx.y][threadIdx.x] = (idx_b < n * n) ? d_b[idx_b] : 0;

        __syncthreads(); // Sync to ensure tiles are fully loaded

        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Store the result if within bounds
    if (row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}