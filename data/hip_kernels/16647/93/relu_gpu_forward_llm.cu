#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x; // Calculate grid stride
    for (int i = tid; i < N; i += gridSize) { // Loop over elements with stride
        out[i] = in[i] > 0 ? in[i] : 0; // Apply ReLU
    }
}