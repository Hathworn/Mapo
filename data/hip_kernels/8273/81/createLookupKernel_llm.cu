#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void createLookupKernel(const int* __restrict__ inds, int total, int* __restrict__ output)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < total)
    {
        int index = inds[idx];
        // Use atomic operation to prevent race conditions
        atomicExch(&output[index], idx);
    }
}