#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate the global position in a single step
    int pos = blockIdx.y * blockDim.y * w + threadIdx.y * w + blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check in a single condition
    if (pos < w * h) {
        image[pos] = value; // Assign the value directly
    }
}