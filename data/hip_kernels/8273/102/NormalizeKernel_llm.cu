#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate 1D index from 2D grid structure for coalesced memory access
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;
   
    if (i < h && j < w) { // Use conditions that are more likely to be true
        // Access memory efficiently with row-major order
        const int pos = i * s + j;
        
        float scale = normalization_factor[pos];
        float invScale = (scale != 0.0f) ? (1.0f / scale) : 1.0f;
        
        image[pos] *= invScale;
    }
}