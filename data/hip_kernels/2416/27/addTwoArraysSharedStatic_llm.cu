#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addTwoArraysSharedStatic(int *v1, int *v2, int *r, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= n) return;

    // Use multiple shared memory buffers
    extern __shared__ int s[];

    int *s_v1 = s;
    int *s_v2 = s + blockDim.x;
    
    // Load data into shared memory from global memory
    s_v1[threadIdx.x] = v1[tid];
    s_v2[threadIdx.x] = v2[tid];
    
    __syncthreads(); // Ensure all data is loaded

    // Compute and store result in global memory
    r[tid] = s_v1[threadIdx.x] + s_v2[threadIdx.x];
}