#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedDynamic(int *a, int *b, int *res, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= n)
    {
        return;
    }

    extern __shared__ int data[];

    int *s_a = data;
    int *s_b = &s_a[blockDim.x];
    // Remove unnecessary third shared memory array
    s_a[threadIdx.x] = a[tid];
    s_b[threadIdx.x] = b[tid];

    // Combine operations and use registers for efficiency
    int temp_res = s_a[threadIdx.x] - s_b[threadIdx.x];
    if (temp_res < 0)
    {
        temp_res = 0;
    }
    
    // Direct assignment to global memory
    res[tid] = temp_res;
}