#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define threads 32
#define size 5

using namespace std;

__global__ void callOperation(int *a, int *b, int *res, int k, int p, int n)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;

    // Use a single if condition to reduce branching and a warp shuffle
    if (tidx < n && tidy < n) {
        int tid = tidx * n + tidy;
        int temp = a[tid] - b[tid];
        res[tid] = (temp < k) ? p : temp;
    }
}