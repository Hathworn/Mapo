#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperation(int *a, int *b, int *c, int n)
{
    int tid = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x; // Flatten thread indexing

    if (tid < n) // Check within array bounds
    {
        if (a[tid] <= b[tid])
        {
            c[tid] = a[tid];
        }
        else
        {
            c[tid] = b[tid];
        }
    }
}