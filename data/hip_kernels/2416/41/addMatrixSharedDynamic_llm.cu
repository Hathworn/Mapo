#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addMatrixSharedDynamic(int *a, int *b, int *res, int n)
{
    // Calculate global thread index
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;

    // Return if thread is out of matrix bounds
    if (tidx >= n || tidy >= n) 
    {
        return;
    }

    // Calculate linear index
    int tid = tidx * n + tidy;
    
    extern __shared__ int arrays[];

    // Assign shared memory pointers
    int *s_a = arrays;
    int *s_b = &arrays[n * n];
    int *s_res = &s_b[n * n];

    // Load data into shared memory
    s_a[tid] = a[tid];
    s_b[tid] = b[tid];
    __syncthreads(); // Ensure all loads completed

    // Perform computation
    s_res[tid] = s_a[tid] + s_b[tid];
    __syncthreads(); // Ensure all computations completed

    // Write result back to global memory
    res[tid] = s_res[tid];
}