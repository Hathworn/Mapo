#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addTwoArrays(int *v1, int *v2, int *r, int n)
{
    // Use shared memory for faster access
    __shared__ int s_v1[blockDim.x];
    __shared__ int s_v2[blockDim.x];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < n) {
        // Load data into shared memory 
        s_v1[threadIdx.x] = v1[tid];
        s_v2[threadIdx.x] = v2[tid];
        __syncthreads();

        // Perform addition using shared memory
        r[tid] = s_v1[threadIdx.x] + s_v2[threadIdx.x];
    }
}