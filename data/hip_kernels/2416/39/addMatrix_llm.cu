#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addMatrix(int *a, int *b, int *res, int n)
{
    // Calculate linear index directly
    int tid = blockDim.x * blockIdx.x + threadIdx.x + 
              (blockDim.y * blockIdx.y + threadIdx.y) * n;

    // Check if within bounds; perform addition if true
    if (tid < n * n) {
        res[tid] = a[tid] + b[tid];
    }
}