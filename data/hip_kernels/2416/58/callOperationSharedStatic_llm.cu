#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedStatic(int *a, int *b, int *c, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= n) {
        return;
    }

    // Declare shared memory with dynamic size using extern keyword
    extern __shared__ int shared_mem[];
    int *s_a = shared_mem;
    int *s_b = shared_mem + blockDim.x;
    int *s_c = shared_mem + 2 * blockDim.x;

    // Use shared memory indices local to the block, not the global index
    int local_id = threadIdx.x;

    s_a[local_id] = a[tid];
    s_b[local_id] = b[tid];

    if (s_a[local_id] <= s_b[local_id])
    {
        s_c[local_id] = s_a[local_id];
    }
    else
    {
        s_c[local_id] = s_b[local_id];
    }
    c[tid] = s_c[local_id];
}