#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedDynamic(int *a, int *b, int *c, int n)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidx >= n || tidy >= n)
    {
        return;
    }

    int tid = tidx * n + tidy;

    extern __shared__ int data[];

    int *s_a = data;
    int *s_b = &s_a[n * n]; // Use 'n' instead of 'size' for clarity
    int *s_c = &s_b[n * n];

    // Load input data into shared memory
    s_a[tid] = a[tid];
    s_b[tid] = b[tid];

    __syncthreads(); // Ensure all threads have loaded data

    // Perform operation and store the result
    if (s_a[tid] >= s_b[tid])
    {
        s_c[tid] = s_a[tid];
    }
    else
    {
        s_c[tid] = s_b[tid];
    }

    __syncthreads(); // Ensure all calculations are done

    // Store result back to global memory
    c[tid] = s_c[tid];
}