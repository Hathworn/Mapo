#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedDynamic(int *a, int *b, int *c, int n)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidx >= n || tidy >= n)
    {
        return;
    }

    int tid = tidx * n + tidy;

    extern __shared__ int data[];

    int *s_a = data;
    int *s_b = &s_a[n * n];  // Use the dynamic shared memory size
    int *s_c = &s_b[n * n];  // Adjust shared memory index

    // Copy to shared memory
    s_a[tid] = a[tid];
    s_b[tid] = b[tid];
    __syncthreads();  // Ensure all threads have loaded data into shared memory

    // Perform operation
    s_c[tid] = min(s_a[tid], s_b[tid]);

    // Write results to global memory
    c[tid] = s_c[tid];
}