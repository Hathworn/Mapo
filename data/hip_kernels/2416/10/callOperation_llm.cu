#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperation(int *a, int *result, int k, int n)
{
    // Calculate thread id in a 1D grid
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;
  
    // Flattened 1D grid index computation
    int tid = tidx * n + tidy;

    // Exit early if out of bounds
    if (tidx >= n || tidy >= n)
    {
        return;
    }

    // Use atomic operation to safely accumulate result
    if (a[tid] == k)
    {
        atomicAdd(result, 1);
    }
}