#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedDynamic(int *a, int *b, int *res, int k, int p, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (tid >= n)
    {
        return;
    }

    extern __shared__ int data[];

    int *s_a = data;
    int *s_b = &s_a[n];

    // Use registers for constants to reduce shared memory accesses
    int s_k = k;
    int s_p = p;

    // Load input data into shared memory
    s_a[threadIdx.x] = a[tid];
    s_b[threadIdx.x] = b[tid];

    // Perform computation using shared memory
    int sum = s_a[threadIdx.x] + s_b[threadIdx.x];
    res[tid] = (sum > s_k) ? s_p : sum;
}