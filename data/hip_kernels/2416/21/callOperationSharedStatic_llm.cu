#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedStatic(int *a, int *b, int *res, int k, int p, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= n) {
        return;
    }

    __shared__ int s_k, s_p;
    extern __shared__ int s_data[];  // Efficient use of shared memory with dynamic allocation
    int *s_a = s_data;
    int *s_b = s_a + blockDim.x;
    int *s_res = s_b + blockDim.x;

    // Load scalar values into shared memory once
    if (threadIdx.x == 0) {
        s_k = k;
        s_p = p;
    }
    __syncthreads();  // Ensure all threads have the scalar values

    // Load array elements into shared memory
    s_a[threadIdx.x] = a[tid];
    s_b[threadIdx.x] = b[tid];
    __syncthreads();  // Ensure all data is loaded before computation

    // Perform computation using shared memory
    s_res[threadIdx.x] = s_a[threadIdx.x] + s_b[threadIdx.x];
    if (s_res[threadIdx.x] > s_k) {
        s_res[threadIdx.x] = s_p;
    }

    // Save results back to global memory
    res[tid] = s_res[threadIdx.x];
}