#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedStatic(int *a, int *b, int x, int *res, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= n)
    {
        return;
    }

    // Utilize shared memory optimally for fewer threads per block
    extern __shared__ int shared_mem[];
    int *s_a = shared_mem;
    int *s_b = shared_mem + blockDim.x;
    int *s_res = shared_mem + 2 * blockDim.x;
    int s_x = x;

    // Ensure that only active threads perform memory operations
    s_a[threadIdx.x] = a[tid];
    s_b[threadIdx.x] = b[tid];

    __syncthreads(); // Synchronize threads in the block before computation

    s_res[threadIdx.x] = ((s_a[threadIdx.x] * s_x) + s_b[threadIdx.x]);

    __syncthreads(); // Synchronize threads before writing results

    res[tid] = s_res[threadIdx.x];
}