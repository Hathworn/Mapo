#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedDynamic(int *a, int *b, int *res, int n)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidx >= n || tidy >= n)
    {
        return;
    }
    int tid = tidx * n + tidy;

    extern __shared__ int data[];

    int *s_a = data;
    int *s_b = &s_a[n * n]; // Corrected shared memory indexing
    int *s_res = &s_b[n * n];

    // Load input data to shared memory
    s_a[tid] = a[tid];
    s_b[tid] = b[tid];

    __syncthreads(); // Ensure all threads have written to shared memory

    // Perform operation
    s_res[tid] = s_a[tid] - s_b[tid];
    if (s_res[tid] < 0)
    {
        s_res[tid] = 0;
    }

    // Store result back to global memory
    res[tid] = s_res[tid];
}