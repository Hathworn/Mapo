#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculateMatrixFormula(int *a, int *b, int *res, int n)
{
    // Calculate global thread ID
    int tid = blockDim.x * blockIdx.x + threadIdx.x + (blockDim.y * blockIdx.y + threadIdx.y) * n;

    // Check if within matrix bounds
    if (tid < n * n) {
        res[tid] = a[tid] - b[tid];
    }
}