#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedStatic(int *a, int *b, int *res, int n)
{
    // Calculate 1D index in the flattened grid
    int tid = blockDim.x * blockIdx.x + threadIdx.x + (blockDim.y * blockIdx.y + threadIdx.y) * n;

    // Check if within bounds
    if (blockIdx.x * blockDim.x + threadIdx.x >= n || blockIdx.y * blockDim.y + threadIdx.y >= n) 
    {
        return;
    }

    // Declare shared memory with dynamic size
    extern __shared__ int shared[];
    int* s_a = shared;
    int* s_b = s_a + n * n;

    // Load data into shared memory
    s_a[tid] = a[tid];
    s_b[tid] = b[tid];
    __syncthreads();  // Ensure all data is loaded before proceeding

    // Perform computation
    int temp_res = s_a[tid] - s_b[tid];
    temp_res = max(temp_res, 0);  // Avoid negative result

    // Write back to global memory
    res[tid] = temp_res;
}