#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use kernel parameters for block and grid indexing
__global__ void add(int *a, int *b, int *sum)
{
    // Calculate unique index for each thread
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Perform addition if index is zero (single-threaded equivalent)
    if (idx == 0) {
        // Utilize shared memory to optimize global memory access
        __shared__ int tempSum;
        tempSum = *a + *b;
        __syncthreads(); // Ensure completion of all ops
        *sum = tempSum;
    }
}