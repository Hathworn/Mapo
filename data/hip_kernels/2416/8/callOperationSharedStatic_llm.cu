#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedStatic(int *a, int *b, int *res, int x, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Exit if thread index is out of bounds
    if (tid >= n)
    {
        return;
    }

    // Use a smaller shared memory size for efficiency
    extern __shared__ int sharedMemory[];
    int *s_a = sharedMemory;
    int *s_b = &s_a[blockDim.x];
    int *s_res = &s_b[blockDim.x];
    
    // Load data into shared memory
    s_a[threadIdx.x] = a[tid];
    s_b[threadIdx.x] = b[tid];

    __syncthreads(); // Ensure all loads are done before computing

    // Perform computation
    s_res[threadIdx.x] = s_a[threadIdx.x] - (s_b[threadIdx.x] * x);

    __syncthreads(); // Ensure computation is complete

    // Write result back to global memory
    res[tid] = s_res[threadIdx.x];
}