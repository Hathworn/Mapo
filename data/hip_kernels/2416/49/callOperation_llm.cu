#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperation(int *a, int *b, int *c, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure thread is within bounds and prevent divergence
    if (tid < n) {
        c[tid] = max(a[tid], b[tid]); // Use max to simplify conditional
    }
}