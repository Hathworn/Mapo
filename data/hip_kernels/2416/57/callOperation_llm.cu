#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperation(int *a, int *b, int *c, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Bounds check to prevent out-of-bounds memory access
    if (tid >= n) return;

    // Use conditional operator to simplify and avoid branch divergence
    c[tid] = (a[tid] <= b[tid]) ? a[tid] : b[tid];
}