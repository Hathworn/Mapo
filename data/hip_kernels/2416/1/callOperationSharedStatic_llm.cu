#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define threads 32
#define size 5

using namespace std;

__global__ void callOperationSharedStatic(int *a, int *b, int *res, int k, int p, int n)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidx >= n || tidy >= n) {
        return;
    }

    int tid = tidx * n + tidy;

    // Use individual shared memory variables instead of arrays for each thread
    __shared__ int s_k, s_p;
    __shared__ int s_a[size * size], s_b[size * size], s_res[size * size];

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        s_k = k;
        s_p = p;
    }

    // Synchronize to ensure shared memory initialization is done
    __syncthreads();

    s_a[tid] = a[tid];
    s_b[tid] = b[tid];
    s_res[tid] = s_a[tid] - s_b[tid];

    if (s_res[tid] < s_k) {
        s_res[tid] = s_p;
    }

    // Write result back to global memory
    res[tid] = s_res[tid];
}