#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addTwoArraysSharedDynamic(int *v1, int *v2, int *r, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimize: Only proceed if tid is within bounds
    if (tid < n)
    {
        extern __shared__ int arrays[];
        int *s_v1 = arrays;
        int *s_v2 = &s_v1[blockDim.x]; // Reduced shared memory usage
        s_v1[threadIdx.x] = v1[tid];  // Use threadIdx.x for shared memory indexing
        s_v2[threadIdx.x] = v2[tid];

        // Perform addition using shared memory
        r[tid] = s_v1[threadIdx.x] + s_v2[threadIdx.x];
    }
}