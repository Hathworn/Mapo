#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedDynamic(int *a, int *b, int *res, int k, int p, int n) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= n) return;

    extern __shared__ int data[]; // Shared memory declaration

    // Pointer initializations for shared memory arrays
    int *s_a = data;
    int *s_b = &s_a[n];
    int *s_res = &s_b[n];

    s_a[tid] = a[tid];  // Load input data to shared memory
    s_b[tid] = b[tid];

    // Directly compute and store results in shared memory
    int temp_res = s_a[tid] - s_b[tid];
    s_res[tid] = (temp_res < k) ? p : temp_res;

    res[tid] = s_res[tid];  // Copy result back to global memory
}