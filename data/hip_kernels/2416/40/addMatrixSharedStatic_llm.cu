#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addMatrixSharedStatic(int *a, int *b, int *res, int n)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidx >= n || tidy >= n)
    {
        return;
    }

    int tid = tidx * n + tidy;

    // Load input data into shared memory with stride pattern
    __shared__ int s_a[BLOCK_SIZE][BLOCK_SIZE], s_b[BLOCK_SIZE][BLOCK_SIZE];

    s_a[threadIdx.x][threadIdx.y] = a[tid];
    s_b[threadIdx.x][threadIdx.y] = b[tid];

    // Synchronize to ensure all data is loaded to shared memory
    __syncthreads();

    // Perform calculation and store result 
    int sum = s_a[threadIdx.x][threadIdx.y] + s_b[threadIdx.x][threadIdx.y];
    res[tid] = sum;
}