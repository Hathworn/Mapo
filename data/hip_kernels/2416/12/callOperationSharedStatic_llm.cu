#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedStatic(int *a, int *res, int x, int n)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidx >= n || tidy >= n) {
        return;
    }

    int tid = tidx * n + tidy;

    extern __shared__ int shared_memory[];  // Use dynamic shared memory
    int* s_a = shared_memory;
    int* s_res = &s_a[n * n];
    int* s_x = &s_res[n * n];

    // Load 'x' only once into shared memory
    *s_x = x;

    // Load input array to shared memory
    s_a[tid] = a[tid];
    __syncthreads();  // Ensure all loads to shared memory are complete

    // Perform computation using shared memory
    s_res[tid] = s_a[tid] * (*s_x);

    // Write back the results
    res[tid] = s_res[tid];
}