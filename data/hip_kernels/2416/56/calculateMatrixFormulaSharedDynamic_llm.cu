#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculateMatrixFormulaSharedDynamic(int *a, int *b, int *res, int n)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidx >= n || tidy >= n) {
        return;
    }

    int tid = tidx * n + tidy;

    extern __shared__ int arrays[];

    int *s_a = arrays;
    int *s_b = &arrays[n * n];

    // Load data into shared memory
    s_a[tid] = a[tid];
    s_b[tid] = b[tid];
    __syncthreads(); // Ensure all data is loaded before proceeding

    // Simple subtraction directly on shared memory
    res[tid] = s_a[tid] - s_b[tid];
}