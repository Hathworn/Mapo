#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedDynamic(int *a, int *b, int *c, int n) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Early return for threads out of range
    if (tid >= n) return;

    extern __shared__ int data[];

    int *s_a = data;
    int *s_b = &s_a[blockDim.x];
    int *s_c = &s_b[blockDim.x];

    // Load a and b into shared memory
    s_a[threadIdx.x] = a[tid];
    s_b[threadIdx.x] = b[tid];

    __syncthreads(); // Ensure all loads to shared memory are complete

    // Perform the operation
    s_c[threadIdx.x] = (s_a[threadIdx.x] >= s_b[threadIdx.x]) ? s_a[threadIdx.x] : s_b[threadIdx.x];

    // Write the result back to global memory
    c[tid] = s_c[threadIdx.x];
}