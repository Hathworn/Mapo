#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedStatic(int *a, int *b, int *res, int k, int p, int n)
{
    // Calculate thread's unique index
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if index is within bounds
    if (idx >= n || idy >= n) return;
    
    int tid = idx * n + idy;

    // Declare shared memory
    extern __shared__ int s_data[];
    int* s_a = s_data;
    int* s_b = s_data + n * n;
    int* s_res = s_data + 2 * n * n;

    // Load data into shared memory
    s_a[tid] = a[tid];
    s_b[tid] = b[tid];
    __syncthreads(); // Ensure all data is loaded

    // Perform operation in shared memory
    int sum = s_a[tid] + s_b[tid];
    s_res[tid] = (sum > k) ? p : sum;

    __syncthreads(); // Ensure all operations are complete

    // Write result back to global memory
    res[tid] = s_res[tid];
}