#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperation(int *a, int *b, int *res, int x, int n) {
    // Use efficient memory access pattern by calculating once and reusing
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        int b_val = b[tid] * x; // Precompute b[tid] * x
        res[tid] = a[tid] - b_val; // Use precomputed value
    }
}