#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedStatic(int *a, int *b, int *c, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if thread ID is within bounds
    if (tid >= n) {
        return;
    }

    // Use stack memory for small fixed-size shared memory allocation
    __shared__ int s_a[1024], s_b[1024];

    // Load values from global to shared memory
    int val_a = a[tid];
    int val_b = b[tid];
    s_a[threadIdx.x] = val_a;
    s_b[threadIdx.x] = val_b;
    __syncthreads(); // Synchronize to ensure all loads are done

    // Perform the operation using the variables from shared memory
    int result = (s_a[threadIdx.x] >= s_b[threadIdx.x]) ? s_a[threadIdx.x] : s_b[threadIdx.x];

    // Write the result back to global memory
    c[tid] = result;
}