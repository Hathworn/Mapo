#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedStatic(int *a, int *b, int *c, int n)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidx >= n || tidy >= n)
    {
        return;
    }

    int tid = tidx * n + tidy;

    // Use a smaller, statically defined shared memory size to optimize resource usage
    __shared__ int s_a[256], s_b[256], s_c[256];

    // Calculate local index for shared memory
    int local_tid = threadIdx.x + blockDim.x * threadIdx.y;

    // Load data into shared memory
    s_a[local_tid] = a[tid];
    s_b[local_tid] = b[tid];
    __syncthreads(); // Ensure all threads have loaded their data

    // Perform operation using shared memory
    s_c[local_tid] = max(s_a[local_tid], s_b[local_tid]);
    __syncthreads(); // Ensure completion of computation

    // Write result back to global memory
    c[tid] = s_c[local_tid];
}