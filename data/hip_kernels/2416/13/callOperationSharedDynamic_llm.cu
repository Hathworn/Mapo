#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedDynamic(int *a, int *res, int x, int n)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidx >= n || tidy >= n) {
        return;
    }

    int tid = tidx * n + tidy;

    extern __shared__ int data[];

    int *s_a = data;
    int *s_res = &s_a[n * n];  // Fix incorrect size calculation

    __shared__ int s_x;
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        s_x = x;  // Optimize by loading x only once
    }
    __syncthreads();  // Ensure s_x is loaded before use

    s_a[tid] = a[tid];
    __syncthreads();  // Ensure all data is loaded before computation

    s_res[tid] = s_a[tid] * s_x;
    res[tid] = s_res[tid];
}