#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperation(int *a, int *b, int *res, int k, int p, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Use block-wide bounds check to reduce divergence
    if (tid < n) {
        int temp = a[tid] + b[tid]; // Cache the sum in a temporary variable

        // Conditional assignment to minimize branch divergence
        res[tid] = (temp > k) ? p : temp;
    }
}