#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedStatic(int *a, int *b, int *c, int n)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidx >= n || tidy >= n)
    {
        return;
    }

    int tid = tidx * n + tidy;

    // Using static shared memory allocation.
    __shared__ int s_a[size];
    __shared__ int s_b[size];

    // Load data into shared memory using each thread.
    s_a[tid] = a[tid];
    s_b[tid] = b[tid];

    // Synchronize threads to ensure all data is loaded.
    __syncthreads();

    // Perform conditional operation directly on shared memory.
    int result = (s_a[tid] <= s_b[tid]) ? s_a[tid] : s_b[tid];

    // Write the result back to global memory.
    c[tid] = result;
}