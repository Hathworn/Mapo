#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define threads 32
#define size 5

using namespace std;

__global__ void callOperationSharedDynamic(int *a, int *b, int *res, int k, int p, int n)
{
    // Calculate the unique thread index
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;
    if (tidx >= n || tidy >= n) return;
    int tid = tidx * n + tidy;

    // Declare shared memory dynamically
    extern __shared__ int data[];
    int *s_a = data;
    int *s_b = &s_a[size * size];
    int *s_res = &s_b[size * size];

    // Load values into shared memory
    s_a[tid] = a[tid];
    s_b[tid] = b[tid];

    // Perform operation and store in shared memory
    s_res[tid] = s_a[tid] - s_b[tid];
    if (s_res[tid] < k) {  // Directly use 'k' as parameter, no need for shared
        s_res[tid] = p;    // Directly use 'p' as parameter
    }

    // Write the result back to global memory
    res[tid] = s_res[tid];
}