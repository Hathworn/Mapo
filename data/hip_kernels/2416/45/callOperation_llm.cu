#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperation(int *niz, int *res, int k, int n)
{
    // Use shared memory to store the value of k for faster access
    extern __shared__ int shared[];
    shared[0] = k;
    __syncthreads();

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Early exit if tid is out of bounds
    if (tid >= n) return;

    // Use the value from shared memory
    if (niz[tid] == shared[0]) {
        atomicAdd(res, 1);
    }
}