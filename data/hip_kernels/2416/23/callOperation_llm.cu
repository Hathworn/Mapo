#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperation(int *a, int *b, int *c, int n)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;

    // Combine boundary checks to reduce branching
    if (tidx < n && tidy < n) 
    {
        int tid = tidx * n + tidy;
        c[tid] = max(a[tid], b[tid]); // Use max function for simplicity
    }
}