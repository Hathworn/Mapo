#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperation(int *a, int *b, int *res, int k, int p, int n) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < n) { // Combine condition check and operation
        int temp = a[tid] - b[tid]; // Use temporary variable to avoid redundant memory accesses
        res[tid] = (temp < k) ? p : temp; // Use ternary operator for concise condition handling
    }
}