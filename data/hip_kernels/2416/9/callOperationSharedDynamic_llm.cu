#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperationSharedDynamic(int *a, int *b, int *res, int x, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= n)
    {
        return;
    }

    extern __shared__ int arrays[];
    
    // Combine data load into shared memory
    int *s_a = arrays;
    int *s_b = &arrays[n];
    s_a[tid] = a[tid];
    s_b[tid] = b[tid];
    
    // Directly operate on memory
    res[tid] = s_a[tid] - (s_b[tid] * x);
}