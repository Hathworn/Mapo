#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperation(int *a, int *b, int *res, int n)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;

    // Combine index calculations and bounds check for efficiency
    if (tidx < n && tidy < n)
    {
        int tid = tidx * n + tidy;
        
        // Inline computation and conditional assignment using max function
        res[tid] = max(0, a[tid] - b[tid]);
    }
}