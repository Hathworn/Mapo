#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperation(int *a, int *b, int *res, int k, int p, int n)
{
    // Optimize thread indexing calculations
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < n && idy < n) {
        int tid = idx * n + idy; // Simplify linear indexing
        int temp_res = a[tid] + b[tid]; // Reduce global memory access
        res[tid] = (temp_res > k) ? p : temp_res; // Use ternary for conditional
    }
}