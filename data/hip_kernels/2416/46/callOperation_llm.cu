#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperation(int *a, int *b, int x, int *res, int n) 
{
    // Use shared memory for faster access
    extern __shared__ int shared_a[];
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < n) {
        shared_a[threadIdx.x] = a[tid]; // Load data into shared memory
        __syncthreads(); // Ensure all threads have loaded data

        // Perform calculation using shared memory
        res[tid] = (shared_a[threadIdx.x] * x) + b[tid];
    }
}