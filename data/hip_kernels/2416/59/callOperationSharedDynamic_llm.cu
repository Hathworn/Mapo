#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedDynamic(int *a, int *b, int *c, int n)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= n) {
        return;
    }

    // Efficient shared memory usage with bank conflict avoidance
    extern __shared__ int data[];

    int *s_a = data;
    int *s_b = &s_a[blockDim.x];
    int *s_c = &s_b[blockDim.x];

    // Load data into shared memory
    s_a[threadIdx.x] = a[tid];
    s_b[threadIdx.x] = b[tid];

    __syncthreads(); // Ensure all data is loaded before proceeding

    // Perform computation using shared memory
    if (s_a[threadIdx.x] <= s_b[threadIdx.x])
    {
        s_c[threadIdx.x] = s_a[threadIdx.x];
    }
    else
    {
        s_c[threadIdx.x] = s_b[threadIdx.x];
    }

    __syncthreads(); // Ensure computation is completed

    // Store the result back to global memory
    c[tid] = s_c[threadIdx.x];
}