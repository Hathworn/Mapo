#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void callOperation(int *a, int *b, int *res, int n)
{
    // Calculate global thread index
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Check if within bounds and compute result
    if (tid < n) 
    {
        int temp = a[tid] - b[tid];  // Calculate subtraction once
        res[tid] = max(temp, 0);     // Use max to avoid conditional
    }
}