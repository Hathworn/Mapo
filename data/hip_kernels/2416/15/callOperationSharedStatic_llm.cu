#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedStatic(int *a, int *b, int *res, int n) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= n) {
        return;
    }

    // Allocation of shared memory
    extern __shared__ int sharedMem[];
    int* s_a = sharedMem;
    int* s_b = &s_a[blockDim.x];
    int* s_res = &s_b[blockDim.x];

    // Load inputs from global to shared memory
    s_a[threadIdx.x] = a[tid];
    s_b[threadIdx.x] = b[tid];
    __syncthreads(); // Ensure all threads have loaded inputs

    // Perform operation
    int local_res = s_a[threadIdx.x] - s_b[threadIdx.x];
    if (local_res < 0) {
        local_res = 0;
    }

    // Write result to shared memory
    s_res[threadIdx.x] = local_res;
    __syncthreads(); // Ensure operation is complete before writing results back

    // Store result back to global memory
    res[tid] = s_res[threadIdx.x];
}