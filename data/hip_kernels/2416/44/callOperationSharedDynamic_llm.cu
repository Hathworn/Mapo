#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedDynamic(int *a, int *b, int *res, int k, int p, int n)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    // Return if index is out of bounds
    if (idx >= n || idy >= n)
    {
        return;
    }

    int tid = idx * n + idy;

    extern __shared__ int data[];

    int *s_a = data;
    int *s_b = &s_a[n * n]; // Correcting the size allocation for shared memory
    int *s_res = &s_b[n * n]; // Improving memory usage calculation

    // Use registers for temporary storage
    int local_k = k;
    int local_p = p;

    // Load data into shared memory
    s_a[tid] = a[tid];
    s_b[tid] = b[tid];

    // Perform operation using shared memory
    int temp_res = s_a[tid] + s_b[tid];

    // Conditional operation
    if (temp_res > local_k)
    {
        temp_res = local_p;
    }

    // Store result
    res[tid] = temp_res;
}