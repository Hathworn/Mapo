#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculateMatrixFormulaSharedStatic(int *a, int *b, int *res, int n)
{
    int tidx = blockDim.x * blockIdx.x + threadIdx.x;
    int tidy = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidx >= n || tidy >= n) {
        return;
    }

    int tid = tidx * n + tidy;

    // Use local variables for temporary storage to make better use of shared memory.
    __shared__ int s_a[size * size];
    __shared__ int s_b[size * size];

    s_a[tid] = a[tid];
    s_b[tid] = b[tid];
    __syncthreads(); // Ensure all threads have loaded data into shared memory

    // Perform calculation directly without additional shared memory usage.
    int result = s_a[tid] - s_b[tid];
    res[tid] = result;
}