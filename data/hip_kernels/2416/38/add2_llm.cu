#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory for intermediate storage and atomic operation for safer updates
__global__ void add2(int a, int b, int *sum)
{
    __shared__ int temp; // Declare shared memory for intermediate sum
    if (threadIdx.x == 0) {
        temp = 0;
    }
    __syncthreads();

    // Only one thread needs to do the addition and update
    if (threadIdx.x == 0) {
        temp = a + b;
        atomicAdd(sum, temp); // Atomic operation to ensure correct update
    }
}