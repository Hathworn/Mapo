#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperation(int *a, int *res, int x, int n)
{
    // Calculate flattened thread ID and ensure within bounds in one step
    int tid = blockDim.x * blockIdx.x + threadIdx.x + 
              (blockDim.y * blockIdx.y + threadIdx.y) * n;
    // Boundary check combined for both grid and thread limits
    if (tid < n * n) {
        res[tid] = a[tid] * x; // Perform the operation
    }
}