#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void callOperationSharedStatic(int *a, int *b, int *res, int k, int p, int n) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid >= n) {
        return;
    }

    // Declare shared memory as extern to avoid fixed size allocation
    extern __shared__ int shared_data[];
    int* s_a = shared_data;
    int* s_b = s_a + n;
    int* s_res = s_b + n;

    // Allocate shared scalars
    __shared__ int s_k, s_p;

    s_k = k; 
    s_p = p;
    s_a[tid] = a[tid];
    s_b[tid] = b[tid];

    // Perform computation using shared memory
    s_res[tid] = s_a[tid] - s_b[tid];
    if (s_res[tid] < s_k) {
        s_res[tid] = s_p;
    }

    // Write result back to global memory
    res[tid] = s_res[tid];
}