#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = hipBlockDim_x * blockIdx.x + threadIdx.x;
    // Replace macro with hipBlockDim_x for better readability
    hiprand_init(seed, tidx, 0, &state[tidx]);
}