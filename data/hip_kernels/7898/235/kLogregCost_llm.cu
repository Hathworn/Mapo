#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Only compute if tx is a valid index
    if (tx < numCases) {
        const int label = static_cast<int>(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];
        
        // Compute logarithm of the predicted probability for the true label
        labelLogProbs[tx] = __logf(labelp);
        
        // Initialize numMax to 0 or 1 based on whether labelp equals maxp
        int numMax = (labelp == maxp) ? 1 : 0;

        // Count occurrences of maxp among all predictions for this case
        for (int i = 0; i < numOut; i++) {
            numMax += (probs[i * numCases + tx] == maxp) && (labelp != maxp);
        }

        // Correct probability calculation
        correctProbs[tx] = (labelp == maxp) ? (1.0f / static_cast<float>(numMax)) : 0.0f;
    }
}