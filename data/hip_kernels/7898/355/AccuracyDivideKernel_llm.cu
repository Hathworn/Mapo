#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use blockIdx.x and threadIdx.x to allow parallel reduction
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        *accuracy /= N; // Only perform division on one thread
    }
}