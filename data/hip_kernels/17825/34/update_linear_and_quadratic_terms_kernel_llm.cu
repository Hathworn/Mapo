#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_linear_and_quadratic_terms_kernel(int32_t n, float old_num_frames, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    float cur_weight = *cur_tot_weight;
    float new_num_frames = old_num_frames + cur_weight;
    float prior_scale_change = 1.0f;

    // Simplify condition check and scale computation
    if (max_count != 0) {
        float old_prior_scale = max(old_num_frames, (float)max_count) / max_count;
        float new_prior_scale = max(new_num_frames, (float)max_count) / max_count;
        prior_scale_change += new_prior_scale - old_prior_scale;
    }

    // Utilize local thread ID variable
    int32_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    for (int32_t i = thread_id; i < n; i += blockDim.x * gridDim.x) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        quadratic[diag_idx] += prior_scale_change;
    }

    // Use unique thread by checking thread and block index
    if (thread_id == 0) {
        linear[0] += prior_offset * prior_scale_change;
    }
}