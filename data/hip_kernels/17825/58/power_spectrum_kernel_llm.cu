#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

    // Calculate start pointers based on blockIdx.x
    const float *Ar = A_in + blockIdx.x * ldi;
    float *Aw = A_out + blockIdx.x * ldo;

    int half_length = row_length / 2;

    // Optimize loop to skip unnecessary iterations
    for (int idx = thread_id; idx < half_length; idx += gridDim.x * blockDim.x) {
        if (idx == 0) continue; // Skip special case handled separately

        float2 val = reinterpret_cast<const float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret);
    }

    // Efficiently process special case using one thread per block
    if (thread_id == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}