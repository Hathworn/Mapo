#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    int stride = blockDim.x; // use blockDim.x for stride
    for (int idx = thread_id; idx < half_length; idx += stride) {
        // ignore special case
        if (idx == 0) continue;
        
        float2 val = reinterpret_cast<float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        if (use_power) {
            Aw[idx] = ret;
        } else {
            Aw[idx] = sqrtf(ret); // consider using fast approximation if precision allows
        }
    }

    // handle special case
    if (thread_id == 0) {
        float real = Ar[0];
        float im = Ar[row_length];
        
        if (use_power) {
            Aw[0] = real * real;
            Aw[half_length] = im * im;
        } else {
            Aw[0] = fabsf(real); // use faster fabsf for float
            Aw[half_length] = fabsf(im);
        }
    }
}