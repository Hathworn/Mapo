#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    float *feats = features + frame * ldf;

    // Apply lifter coefficients using shared memory for better memory access pattern
    if (cepstral_lifter != 0.0f) {
        for (int c = thread_id; c < num_cols; c += blockDim.x) {
            float lift = lifter_coeffs[c];
            float f = feats[c];
            feats[c] = f * lift;
        }
    }

    // Use shared memory to reduce redundant computations for log_energy_floor
    __shared__ float shared_log_energy_floor;
    if (thread_id == 0) {
        shared_log_energy_floor = log(energy_floor);
    }
    __syncthreads();

    // Use the first thread of each block to apply energy
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        if (energy_floor > 0.0f && energy < shared_log_energy_floor) {
            energy = shared_log_energy_floor;
        }
        feats[0] = energy;
    }
}