#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void add(int n, float* a, float* b, float* sum) {
    // Optimizing with shared memory use
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    __shared__ float s_a[1024]; // Shared memory allocation
    __shared__ float s_b[1024]; // Shared memory allocation
    if (index < n) {
        s_a[threadIdx.x] = a[index]; // Load into shared memory
        s_b[threadIdx.x] = b[index]; // Load into shared memory
        __syncthreads(); // Synchronize threads before use
        
        for (int i = index; i < n; i += stride) {
            sum[i] = s_a[threadIdx.x] + s_b[threadIdx.x]; // Use shared memory
        }
    }
}