#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cube(float *d_out, const float *d_in) {
    // Use blockIdx.x to support arrays larger than block size
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure index doesn't exceed the bounds of array
    if (idx < N) { 
        float f = d_in[idx];
        d_out[idx] = f * f * f;
    }
}