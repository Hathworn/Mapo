#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/**
* calculate pi
*/

// For the CUDA runtime routines (prefixed with "cuda_")

#define NUMTHREADS 10240
#define ITERATIONS 1e12

/**
* CUDA Kernel Device code
*
*/

/******************************************************************************
* Host main routine
*/
__global__ void calculatePi(double *piTotal, long int iterations, int totalThreads)
{
    long int i;
    double piPartial = 0.0;

    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate initial and end iteration for each thread
    long int chunk_size = iterations / totalThreads;  // Divided chunk for each thread
    long int initialIteration = chunk_size * index;
    long int endIteration = initialIteration + chunk_size;

    // Calculate partial Pi
    for (i = initialIteration; i < endIteration; i += 2) {
        piPartial += (4.0 / (2 * i + 1)) - (4.0 / (2 * i + 3));
    }

    // Store partial result in global memory
    piTotal[index] = piPartial;

    // Ensure all threads have written their partial results
    __syncthreads();

    // Accumulate results in the first thread
    if (index == 0) {
        for (i = 1; i < totalThreads; i++) {
            piTotal[0] += piTotal[i];
        }
    }
}