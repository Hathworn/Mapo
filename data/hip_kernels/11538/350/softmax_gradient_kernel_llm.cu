#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    // Point to data slice for this block
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    
    // Efficient reduction using a single atomic addition
    float tmp = 0.0f;
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    
    atomicAdd(&reduction_buffer[0], tmp);
    __syncthreads();
    
    // Read reduced result from shared memory
    tmp = reduction_buffer[0];
    
    // Compute gradient
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}