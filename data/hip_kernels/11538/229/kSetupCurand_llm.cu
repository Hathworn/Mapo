#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate global thread index for unique sequence
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    // Each thread gets a different sequence number
    hiprand_init(seed, tidx, 0, &state[tidx]);
}