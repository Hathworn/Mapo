#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii /= d3;
    z = ii % d2;
    ii /= d2;
    y = ii % d1;
    ii /= d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii /= d3;
    z = ii % d2;
    ii /= d2;
    y = ii % d1;
    ii /= d1;
    x = ii;
    w /= scale_factor;
    z /= scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x 
            + threadIdx.y * blockDim.y * gridDim.x * blockDim.x * blockIdx.y;
    if (ii >= no_elements) return;

    // Use temporary variable to reduce redundant indexing
    float sum = 0.0f;
    for (int i = 0; i < scale_factor; i++) {
        for (int j = 0; j < scale_factor; j++) {
            int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
            sum += gradOutput_data[ipidx]; // Accumulate in temporary variable
        }
    }
    gradInput_data[ii] = sum; // Single write operation
}