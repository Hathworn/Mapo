#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use atomic operation to ensure thread safety when updating accuracy
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        atomicExch(accuracy, *accuracy / N);
    }
}