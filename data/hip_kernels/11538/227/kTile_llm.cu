#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate unique thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x; // Optimize thread stride calculation

    for (uint i = idx; i < tgtWidth * tgtHeight; i += stride) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        tgt[i] = src[srcY * srcWidth + srcX]; // Efficient memory access pattern
    }
}