#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;  // Initialize a local storage for reduction

    // Ensure valid indices and accumulate dot product locally
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        sum += a[eidx] * b[eidx];
    }

    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Unrolling reduction within a block
    for (uint stride = DP_BLOCKSIZE / 2; stride > 32; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}