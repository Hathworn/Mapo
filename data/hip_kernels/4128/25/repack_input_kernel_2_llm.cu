#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void repack_input_kernel_2(float *input, float *re_packed_input, int w, int h, int c)
{
    // Optimize memory access by using shared memory
    __shared__ float shared_input[32];

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    const int items_per_channel = w * h;

    int c_pack = index % 32;
    int chan_index = index / 32;
    int chan = (chan_index * 32) % c;
    int i = (chan_index * 32) / c;

    if (i < items_per_channel)
    {
        // Load data into shared memory
        shared_input[c_pack] = input[(chan + c_pack) * items_per_channel + i];
        __syncthreads(); // Ensure all threads load their data

        // Write from shared memory to global memory
        re_packed_input[chan * items_per_channel + i * 32 + c_pack] = shared_input[c_pack];
    }
}