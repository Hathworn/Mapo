#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_uint32_kernel_2(uint32_t *src, uint32_t *dst, int src_h, int src_w, int src_align, int dst_align)
{
    // Allocate shared memory for the tile and ensure alignment
    __shared__ uint32_t tmp[32][33]; // Use 2D array for better cache locality
    const int w_align = 33;

    // Calculate the aligned width to handle memory correctly
    const int src_w_align = (src_w + 31) & ~31; // Use bitwise to achieve alignment
    const int warps_in_width = src_w_align / 32;

    // Determine the local position of the thread within the block
    const int local_x = threadIdx.x & 31; // Use bitwise to reduce instruction count
    const int local_y = threadIdx.x >> 5; // Decompose threadIdx.x using bitwise

    // Determine the global position on the grid
    const int global_index = blockIdx.x;
    const int global_x_index = global_index % warps_in_width;
    const int global_y_index = global_index / warps_in_width;

    const int global_x = (global_x_index << 5) + local_x; // Use shift for multiplication by 32
    const int global_y = (global_y_index << 5) + local_y;

    // Load src into shared memory with boundary checks
    if (global_x < src_w && global_y < src_h) {
        tmp[local_y][local_x] = src[global_y * src_align + global_x];
    }
    __syncthreads();

    // Perform the transpose within shared memory
    const int new_global_x = (global_y_index << 5) + local_x;
    const int new_global_y = (global_x_index << 5) + local_y;

    // Store result back to the destination with boundary checks
    if (new_global_x < src_h && new_global_y < src_w) {
        dst[new_global_y * (dst_align / 32) + new_global_x] = tmp[local_x][local_y];
    }
}