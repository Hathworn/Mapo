#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calc_avg_activation_kernel(float *src, float *dst, int size, int channels, int batches)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size * batches) {
        int xy = i % size;
        int b = i / size;
        int base = size * channels * b;

        // Use float accumulator to minimize precision loss
        float sum = 0.0f;
        for (int c = 0; c < channels; ++c) {
            sum += src[xy + size * c + base];
        }
        dst[i] = sum / channels;
    }
}