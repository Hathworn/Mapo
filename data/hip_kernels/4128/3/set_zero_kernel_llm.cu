#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_zero_kernel(float *src, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for better utilization
    for (; i < size; i += blockDim.x * gridDim.x)
    {
        src[i] = 0;
    }
}