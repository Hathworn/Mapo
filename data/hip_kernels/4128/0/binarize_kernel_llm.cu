#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Calculate thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process multiple elements per thread using stride loop
    int stride = blockDim.x * gridDim.x;
    while (i < n) {
        binary[i] = (x[i] >= 0) ? 1 : -1;
        i += stride;
    }
}