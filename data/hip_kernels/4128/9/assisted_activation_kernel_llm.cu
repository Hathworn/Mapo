#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved assisted_activation_kernel function.
__global__ void assisted_activation_kernel(float alpha, float *output, float *gt_gpu, float *a_avg_gpu, int size, int channels, int batches)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int b = i / size;

    // Process only valid batches.
    if (b < batches) {
        int xy = i % size;
        
        // Precompute index outside the loop to save redundant calculations.
        int index_base = size * channels * b + xy;

        // Optimize loop by reducing array access computation.
        for (int c = 0; c < channels; ++c) {
            int output_index = index_base + size * c;
            
            // Use atomic operations to safely update output in parallel.
            atomicAdd(&output[output_index], alpha * gt_gpu[i] * a_avg_gpu[i]);
        }
    }
}