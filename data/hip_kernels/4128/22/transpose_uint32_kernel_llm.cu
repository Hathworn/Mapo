#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_uint32_kernel(uint32_t *src, uint32_t *dst, int src_h, int src_w, int src_align, int dst_align)
{
    // Calculate global 1D index for the current thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate row (i) and column (j) indices based on the global index
    int i = index % src_h;
    int j = index / src_h;

    // Ensure that column index (j) is within bounds
    if (j < src_w)
    {
        // Perform the transpose operation
        dst[j * dst_align / 32 + i] = src[i * src_align + j];
    }
}