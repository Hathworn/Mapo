#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n * c) return;  // Simplified index calculation for combined (n, c) dimension

    int k = id % c;
    int b = id / c;  // Remove redundant division

    int out_index = k + c * b;
    float sum = 0.0f;  // Use local variable for summation

    // Use temporary variable to hold area size
    int area = w * h;
    int idx_offset = h * w * (k + b * c);
    for (int i = 0; i < area; ++i) {
        sum += input[i + idx_offset];  // Reduce repeated arithmetic
    }

    output[out_index] = sum / area;  // Assign computed average
}