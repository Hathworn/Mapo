#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void assisted_activation2_kernel(float alpha, float *output, float *gt_gpu, float *a_avg_gpu, int size, int channels, int batches)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Derive batch and spatial position
    int xy = i % size;
    int b = i / size;
    float beta = 1 - alpha;

    // Check if current batch is valid
    if (b < batches) {
        // Use shared memory if possible for reduction in global memory accesses
        __shared__ float shared_output[BLOCK_SIZE];
        
        // Iterate over channels
        for (int c = 0; c < channels; ++c) {
            if (gt_gpu[i] == 0)
                shared_output[threadIdx.x] = output[xy + size*(c + channels*b)] * beta;
        }
        
        // Sync threads before writing back to global memory
        __syncthreads();
        
        // Write processed values back to output
        for (int c = 0; c < channels; ++c) {
            output[xy + size*(c + channels*b)] = shared_output[threadIdx.x];
        }
    }
}
```
