#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x;
    if (f >= n) return;

    extern __shared__ float shared_weights[]; // Use shared memory
    float mean = 0.0f;

    // Load weights to shared memory
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[i] = fabs(weights[f*size + i]);
    }
    __syncthreads();

    // Calculate mean
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        mean += shared_weights[i];
    }
    __syncthreads();

    // Reduce mean within a block
    if (threadIdx.x == 0) {
        for (int i = 1; i < blockDim.x; i++) {
            mean += shared_weights[i];
        }
        mean = mean / size;
    }
    __syncthreads();

    // Broadcast the mean value
    mean = shared_weights[0];

    // Binarize weights
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
    }
}