#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_mean_kernel(float *weights, int n, int size, float *binary, float *mean_arr_gpu)
{
    // Optimize thread index calculations for improved readability
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n * size) return; // Check boundary in one step

    // Determine filter index
    int f = i / size;

    // Retrieve and apply mean to binarize weights
    float mean = mean_arr_gpu[f];
    binary[i] = (weights[i] > 0) ? mean : -mean;
}