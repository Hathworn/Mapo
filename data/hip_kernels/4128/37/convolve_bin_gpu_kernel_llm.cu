#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __host__ static inline uint8_t xnor_bit1(uint8_t a, uint8_t b) {
    return ~(a^b) & 0b1;
}

__device__ __host__ static inline unsigned char get_bit(unsigned char const* const src, size_t index) {
    size_t src_i = index / 8;
    int src_shift = index % 8;
    unsigned char val = (src[src_i] & (1 << src_shift)) > 0;
    return val;
}

__global__ void convolve_bin_gpu_kernel(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n, int size, int pad, int new_lda, float *mean_arr_gpu)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    int x = index % in_w;
    int index2 = index / in_w;
    int y = index2 % in_h;
    int fil = index2 / in_h;

    __shared__ uint32_t weights_shared[3*3*1024*6/32 + 1];
    __shared__ uint32_t input_shared[416*416/32 + 1];

    int weights_size = size*size*in_c / 32 + 1;
    int input_shared_size = in_w*in_h / 32 + 1;

    int min_index = blockIdx.x*blockDim.x;
    int min_fil = (min_index / in_w) / in_h;
    int max_index = (blockIdx.x+1)*blockDim.x - 1;
    int max_fil = (max_index / in_w) / in_h;

    for (int tmp_fil = min_fil; tmp_fil <= max_fil; tmp_fil++) {
        for (int s = threadIdx.x; s < weights_size; s += blockDim.x) {
            weights_shared[s + (tmp_fil - min_fil)*new_lda/32] = ((uint32_t *)weights)[tmp_fil*new_lda / 32 + s];
        }
    }
    __syncthreads();  // Sync threads after loading weights into shared memory

    for (int chan = 0; chan < in_c; ++chan)
    {
        int input_pre_index = chan*in_w*in_h;
        int add_input_index = input_pre_index % 32;

        for (int s = threadIdx.x; s < input_shared_size; s += blockDim.x) {
            input_shared[s] = ((uint32_t *)input)[input_pre_index / 32 + s];
        }
        __syncthreads();  // Sync threads after loading input into shared memory

        if (fil < n) 
        {
            int sum = 0;
            int good_val = 0;
            int output_index = fil*in_w*in_h + y*in_w + x;
            
            for (int f_y = 0; f_y < size; ++f_y) {
                int input_y = y + f_y - pad;
                if (input_y < 0 || input_y >= in_h) continue;  // Skip invalid indices

                for (int f_x = 0; f_x < size; ++f_x) {
                    int input_x = x + f_x - pad;
                    if (input_x < 0 || input_x >= in_w) continue;  // Skip invalid indices

                    int weights_shared_index = (fil - min_fil)*new_lda + chan*size*size + f_y*size + f_x;
                    uint8_t w_bit = get_bit((uint8_t *)weights_shared, weights_shared_index);

                    int input_shared_index = input_y*in_w + input_x + add_input_index;
                    uint8_t in_bit = get_bit((uint8_t *)input_shared, input_shared_index);

                    int res = xnor_bit1(in_bit, w_bit);
                    sum += res;
                    good_val++;
                }
            }
            sum = sum - (good_val - sum);
            atomicAdd(&output[output_index], sum * mean_arr_gpu[fil]);
        }
    }
}