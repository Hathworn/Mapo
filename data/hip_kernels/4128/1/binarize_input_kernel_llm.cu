#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;

    // Compute mean using shared memory
    extern __shared__ float shared_input[];

    float local_mean = 0;
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        shared_input[threadIdx.x] = fabs(input[i * size + s]);
        local_mean += shared_input[threadIdx.x];
    }

    // Reduce sum in shared memory
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
        __syncthreads();  // Synchronization barrier
        if (threadIdx.x < offset) {
            shared_input[threadIdx.x] += shared_input[threadIdx.x + offset];
        }
    }

    // Mean computation using the sum stored in shared memory
    __syncthreads();  // Ensure all threads are done computing
    local_mean = shared_input[0];
    local_mean /= n;

    // Binarize input based on computed mean
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? local_mean : -local_mean;
    }
}