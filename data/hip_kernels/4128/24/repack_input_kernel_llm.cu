#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void repack_input_kernel(float *input, float *re_packed_input, int w, int h, int c)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int items_per_channel = w * h;
    const int pack_size = 32;

    // Simplified channel index calculation
    int total_pack_size = (c + pack_size - 1) / pack_size * pack_size; // Align to pack size
    if (index >= total_pack_size * items_per_channel) return; // Boundary check

    int i = index / total_pack_size;   // Item index
    int chan_pack_offset = index % total_pack_size; // Offset within the pack
    int chan_index = chan_pack_offset / pack_size * pack_size; // Base channel index
    int c_pack = chan_pack_offset % pack_size; // c_pack within this offset section

    if (chan_index + c_pack < c) { // Valid channel check
        float src = input[(chan_index + c_pack) * items_per_channel + i];
        re_packed_input[chan_index * items_per_channel + i * pack_size + c_pack] = src;
    }
}