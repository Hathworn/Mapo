#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc2(float *Ad, float *Bd) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx < 1024) {
        float a_value = Ad[tx]; // Cache the value of Ad[tx]
        for (int i = 0; i < ITER; i++) {
            a_value += 1.0f; // Increment cached value
            for (int j = 0; j < 256; j++) {
                Bd[tx] = a_value; // Use cached value for Bd[tx]
            }
        }
        Ad[tx] = a_value; // Write back the cached value to global memory
    }
}