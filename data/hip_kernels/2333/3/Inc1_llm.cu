#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Inc1(float *Ad, float *Bd) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx < 1) {
        float aTemp = Ad[tx]; // Load once, use temporary variable
        for (int i = 0; i < ITER; i++) {
            aTemp = aTemp + 1.0f; // Update temporary variable
            float bTemp = aTemp;  // Use temporary variable for inner loop
            for (int j = 0; j < 256; j++) {
                Bd[tx] = bTemp; // Reduce redundant writes
            }
        }
        Ad[tx] = aTemp; // Write back only once
    }
}