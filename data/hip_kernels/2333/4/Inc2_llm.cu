#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Inc2(float *Ad, float *Bd) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx < 1024) {
        for (int i = 0; i < ITER; i++) {
            float temp = Ad[tx] + 1.0f; // Use temporary variable to avoid multiple memory access
            Ad[tx] = temp;
            Bd[tx] = temp; // Minimize write operations to Bd
        }
    }
}