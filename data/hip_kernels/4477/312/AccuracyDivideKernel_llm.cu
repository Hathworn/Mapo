#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Launch only one thread; avoid unnecessary parallelism
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}