#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_priority(int *new_priority, int n_selected, const int *new_idx, int n_ws, const int *idx, const int *priority) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n_selected) {
        int my_new_idx = new_idx[tid];
        // Use shared memory to reduce global memory access
        extern __shared__ int shared_idx[];
        extern __shared__ int shared_priority[];

        // Each thread loads data into shared memory
        int i = threadIdx.x;
        if (i < n_ws) {
            shared_idx[i] = idx[i];
            shared_priority[i] = priority[i];
        }
        __syncthreads();

        // Loop through the shared memory arrays
        for (int j = 0; j < n_ws; j++) {
            if (shared_idx[j] == my_new_idx) {
                new_priority[tid] = shared_priority[j] + 1;
                break; // Exit the loop early once a match is found
            }
        }
    }
}