#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nan_kernel(float* data, const bool* mask, int len, float nan) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Use warp divergence minimization by iterating over all threads in a block
    for (int i = tid; i < len; i += blockDim.x * gridDim.x) {
        if (!mask[i]) data[i] = nan;
    }
}