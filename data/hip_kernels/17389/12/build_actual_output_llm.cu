#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void build_actual_output(int *output, int n_rows, int k, const int *idx_labels, const int64_t *indices) {
    int element = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Early exit for out-of-bounds threads
    if (element < n_rows * k) {
        int ind = indices[element];
        output[element] = idx_labels[ind];
    }
}