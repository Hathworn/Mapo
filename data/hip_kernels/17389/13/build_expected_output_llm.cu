#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access by using shared memory for labels
__global__ void build_expected_output(int *output, int n_rows, int k, const int *labels) {
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    if (row >= n_rows) return;

    int cur_label = labels[row];
    for (int i = threadIdx.y; i < k; i += blockDim.y) {
        output[row * k + i] = cur_label;
    }
}