#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void optimizedHistKernel(int* bins, int nbins, int* in, int nrows) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    auto offset = blockIdx.y * nrows;
    auto binOffset = blockIdx.y * nbins;

    // Use shared memory for bins to reduce global memory accesses
    extern __shared__ int sharedBins[];
    for (int i = threadIdx.x; i < nbins; i += blockDim.x) {
        sharedBins[i] = 0; // Initialize shared bins
    }
    __syncthreads();

    for (; tid < nrows; tid += stride) {
        int id = in[offset + tid];
        if (id < 0)
            id = 0;
        else if (id >= nbins)
            id = nbins - 1;
        atomicAdd(&sharedBins[id], 1); // Atomic add in shared memory
    }
    __syncthreads();

    // Commit results from shared memory to global memory
    for (int i = threadIdx.x; i < nbins; i += blockDim.x) {
        atomicAdd(bins + binOffset + i, sharedBins[i]);
    }
}