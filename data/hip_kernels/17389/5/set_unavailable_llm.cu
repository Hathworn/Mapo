#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_unavailable(bool *available, int n_rows, const int *idx, int n_selected) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Process multiple elements per thread to reduce block divergence
    for (int i = tid; i < n_selected; i += stride) {
        available[idx[i]] = false;
    }
}