#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void to_float(float *out, int *in, int size) {
    // Calculate global thread ID
    int element = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop to handle large arrays
    for (int i = element; i < size; i += gridDim.x * blockDim.x) {
        out[i] = static_cast<float>(in[i]); // Use explicit cast
    }
}