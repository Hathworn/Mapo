#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void postprocess_matrix(float* matrix, long* long_indices, int* indices, unsigned int N_POINTS, unsigned int K)
{
    // Calculate the global thread ID
    unsigned int TID = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Bounds check
    if (TID < N_POINTS * K) {
        // Directly cast long index to int
        indices[TID] = static_cast<int>(long_indices[TID]);
    }
}