#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < size) {
        r[i] = x[i] * y[i] + z[i]; // Perform computation if within bounds
    }
}