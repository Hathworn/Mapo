#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void worker(double * a, long n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        // Use register variable for frequently used value to minimize memory access
        double val = a[i];
        a[i] = val + i;
    }
}