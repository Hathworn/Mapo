#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initRandomizer(unsigned int seed, hiprandState* state){
int idx = blockIdx.x * blockDim.x + threadIdx.x;
hiprand_init(seed, idx, 0, &state[idx]);
}