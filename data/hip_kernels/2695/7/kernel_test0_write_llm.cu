#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    unsigned int* orig_ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* ptr = orig_ptr;

    // Early exit if the block's starting pointer exceeds the end pointer
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);
    unsigned int pattern = 1;
    unsigned long mask = 4;

    *ptr = pattern;

    // Loop processing, optimized with fewer operations
    while (true) {
        ptr = (unsigned int*) (((unsigned long)orig_ptr) | mask);

        if (ptr >= block_end) {
            break;
        }

        *ptr = pattern;
        
        // Optimize pattern and mask updates
        pattern <<= 1;
        mask <<= 1;

        if (ptr == orig_ptr) {
            continue;
        }
    }
}