#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Terminate early if the start of this block is beyond the end pointer
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Calculate number of iterations needed, considering the end of data segment
    unsigned int iterCount = min(BLOCKSIZE / sizeof(unsigned int), 
                                 ((unsigned int*)end_ptr - ptr));

    unsigned int k = offset;
    unsigned pat = pattern;
    
    // Loop to write pattern as per logic
    for (unsigned int i = 0; i < iterCount; i++) {
        ptr[i] = pat;
        k++;

        // Use ternary operator for concise condition handling
        pat = (k >= 32) ? lb : ((pat << 1) | sval);
        k = (k >= 32) ? 0 : k;
    }
}