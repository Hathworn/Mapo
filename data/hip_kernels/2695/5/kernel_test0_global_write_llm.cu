#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int pattern = 1;

    unsigned long mask = 4;

    *ptr = pattern;

    while (ptr < end_ptr) {
        // Calculate the next address using mask and optimize bit manipulation
        unsigned int* next_ptr = (unsigned int*)((unsigned long)_ptr | mask);
        // If mask doesn't change the address, double the mask
        while (next_ptr == ptr && next_ptr < end_ptr) {
            mask <<= 1;
            next_ptr = (unsigned int*)((unsigned long)_ptr | mask);
        }
        // Check if the next address is outside the end_ptr
        if (next_ptr >= end_ptr) {
            break;
        }
        // Write pattern to next address
        *next_ptr = pattern;
        
        // Update state for next iteration
        ptr = next_ptr;
        pattern <<= 1;
        mask <<= 1;
    }
}