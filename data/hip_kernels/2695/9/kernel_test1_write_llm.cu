#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    // Calculate the starting position for this block
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Quickly check if this block is out of bounds
    if (ptr >= (unsigned long*) end_ptr) {
        return;
    }

    // Use idx to eliminate recalculation of address offset
    unsigned int idx = threadIdx.x;
    int stride = blockDim.x;

    // Loop with thread-stride to maximize parallel execution
    for (unsigned int i = idx; i < BLOCKSIZE / sizeof(unsigned long); i += stride){
        ptr[i] = (unsigned long) &ptr[i];
    }
    
    return;
}