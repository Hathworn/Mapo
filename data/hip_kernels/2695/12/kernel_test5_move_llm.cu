#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCKSIZE 256 // Update as needed

__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);

    // Check if the pointer is out of bounds
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    unsigned int half_count = BLOCKSIZE / sizeof(unsigned int) / 2;
    unsigned int* ptr_mid = ptr + half_count;

    // Use shared memory for optimization
    __shared__ unsigned int shared_memory[BLOCKSIZE / sizeof(unsigned int)];

    // Load data into shared memory
    for (unsigned int i = 0; i < half_count; i++) {
        shared_memory[i] = ptr[i];
    }

    __syncthreads(); // Sync before using shared memory data

    // First transformation
    for (unsigned int i = 0; i < half_count - 8; i++) {
        ptr[i + 8] = shared_memory[i];
    }

    // Second transformation
    for (unsigned int i = 0; i < 8; i++) {
        ptr[i] = shared_memory[half_count - 8 + i];
    }

    return;
}