#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    unsigned int i;
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);

    // Check boundary once, no iteration needed if out of bounds
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Use a single loop with conditionals to reduce iterations
    for (i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        if (i % MOD_SZ == offset) {
            ptr[i] = p1;  // Set to p1 when condition met
        } else {
            ptr[i] = p2;  // Set to p2 otherwise
        }
    }

    return;
}