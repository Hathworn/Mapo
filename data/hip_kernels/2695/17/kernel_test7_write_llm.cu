#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    // Calculate the global thread index
    unsigned int idx = blockIdx.x * BLOCKSIZE + threadIdx.x * sizeof(unsigned int);

    // Pointer to the location to be written
    unsigned int* ptr = (unsigned int*) (_ptr + idx);
    unsigned int* start_ptr = (unsigned int*) _start_ptr;

    // Early exit if the calculated pointer is beyond the end pointer
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Optimized writing using threads
    unsigned int i = threadIdx.x;
    if (i < BLOCKSIZE / sizeof(unsigned int)) {
        ptr[i] = start_ptr[i];
    }

    return;
}