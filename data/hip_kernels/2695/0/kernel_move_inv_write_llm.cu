#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    if (ptr >= (unsigned int*) end_ptr) { 
        return;
    }

    // Use loop unrolling to optimize memory writing
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        ptr[i] = pattern;
    }
}