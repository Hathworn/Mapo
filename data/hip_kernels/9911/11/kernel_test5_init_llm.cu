#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test5_init(char* _ptr, char* end_ptr)
{
    // Optimize the loop to reduce operations
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int p1 = 1;
    unsigned int p2;
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i += 16) {
        p2 = ~p1;

        // Use loop unrolling to reduce redundant code
        #pragma unroll
        for (int j = 0; j < 16; j += 2) {
            ptr[i + j]     = (j % 4 < 2) ? p1 : p2;
            ptr[i + j + 1] = (j % 4 < 2) ? p1 : p2;
        }

        // Update p1 efficiently outside the inner loop
        p1 <<= 1;
        if (p1 == 0) {
            p1 = 1;
        }
    }

    return;
}