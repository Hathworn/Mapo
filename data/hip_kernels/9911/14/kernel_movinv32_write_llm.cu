#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);
    unsigned int idx = threadIdx.x;

    // Early exit if out of bounds
    if (ptr >= (unsigned int*) end_ptr) return;
    
    unsigned int k = offset + idx; // Calculate k based on thread index
    unsigned int pat = pattern;
    
    // If current k is within the range, perform shift directly
    if (k >= 32) {
        k = 0;
        pat = lb;
    } else {
        pat = pat << k; // Calculate pattern shift only once per thread
    }

    unsigned int num_iter = BLOCKSIZE / sizeof(unsigned int);
    for (unsigned int i = idx; i < num_iter; i += blockDim.x) {
        ptr[i] = pat;
        k++;
        if (k >= 32) {
            k = 0;
            pat = lb;
        } else {
            pat = (pat << 1) | sval; // Shift pattern and append sval
        }
    }
}