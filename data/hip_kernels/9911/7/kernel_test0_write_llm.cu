#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    unsigned int* orig_ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* ptr = orig_ptr;
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);

    unsigned int pattern = 1;
    unsigned long mask = 4;

    *ptr = pattern;

    // Use shared memory for mask to reduce register pressure
    __shared__ unsigned long shared_mask;
    shared_mask = mask;

    // Loop through blocks for calculation
    while (ptr < block_end)
    {
        ptr = (unsigned int*)((unsigned long)orig_ptr | shared_mask);
        if (ptr == orig_ptr) {
            shared_mask = shared_mask << 1;
            continue;
        }
        if (ptr >= block_end) {
            break;
        }

        *ptr = pattern;
        pattern = pattern << 1;
        shared_mask = shared_mask << 1;  // Update shared mask
    }
    return;
}