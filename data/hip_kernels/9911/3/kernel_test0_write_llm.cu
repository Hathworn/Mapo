#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCKSIZE 256 // Assuming BLOCKSIZE is defined somewhere

__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);

    // Avoid unnecessary processing for threads with out-of-bound addresses
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Precompute the number of elements to be written in this block
    unsigned int numElements = BLOCKSIZE / sizeof(unsigned int);

    // Optimize with the intrinsic `memset` for faster writes if possible
    // Otherwise, continue using the loop for setting pattern
    #pragma unroll
    for (unsigned int i = 0; i < numElements; i++) {
        ptr[i] = pattern;
    }
}