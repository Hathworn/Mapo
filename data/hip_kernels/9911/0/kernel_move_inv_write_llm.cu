#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Check one time after calculating the memory position
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Use a single int for iteration
    int tid = threadIdx.x;
    int stride = blockDim.x;

    // Iterate over block using threads for potentially better parallelism
    for (unsigned int i = tid; i < BLOCKSIZE / sizeof(unsigned int); i += stride) {
        ptr[i] = pattern;
    }
}