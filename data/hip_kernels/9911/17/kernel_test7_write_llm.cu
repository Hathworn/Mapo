#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* start_ptr = (unsigned int*) _start_ptr;
    
    // Early exit if out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Use thread index for parallel copying
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        ptr[i] = start_ptr[i];
    }
}