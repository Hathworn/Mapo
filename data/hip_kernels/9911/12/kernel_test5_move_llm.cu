#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Early exit if pointer exceeds end_ptr
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int half_count = BLOCKSIZE / sizeof(unsigned int) / 2;
    unsigned int* ptr_mid = ptr + half_count;

    // Use shared memory to reduce global memory operations
    __shared__ unsigned int shared_mem[BLOCKSIZE / sizeof(unsigned int)];

    // Load memory into shared memory
    for (unsigned int i = threadIdx.x; i < half_count; i += blockDim.x) {
        shared_mem[i] = ptr[i];
    }
    
    __syncthreads();

    // Copy data to the middle section using shared memory
    for (unsigned int i = threadIdx.x; i < half_count; i += blockDim.x) {
        ptr_mid[i] = shared_mem[i];
    }
    
    __syncthreads();

    // Move first part to later positions using shared memory
    for (unsigned int i = threadIdx.x; i < half_count - 8; i += blockDim.x) {
        ptr[i + 8] = shared_mem[i];
    }
    
    // Move last 8 elements
    if (threadIdx.x < 8) {
        ptr[threadIdx.x] = ptr_mid[half_count - 8 + threadIdx.x];
    }
}