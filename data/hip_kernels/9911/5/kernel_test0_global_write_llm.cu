#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    // Use shared memory to reduce global memory accesses.
    __shared__ unsigned int shared_pattern;

    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int* orig_ptr = ptr;

    // Initialize the pattern in shared memory once.
    shared_pattern = 1;
    __syncthreads(); // Ensure all threads have the initialized pattern

    unsigned long mask = 4;

    *ptr = shared_pattern; // Write pattern from shared memory

    while(ptr < end_ptr){

        ptr = (unsigned int*) ( ((unsigned long)orig_ptr) | mask);
        if (ptr == orig_ptr){
            mask = mask << 1;
            continue;
        }
        if (ptr >= end_ptr){
            break;
        }

        // Write pattern from shared memory
        *ptr = shared_pattern;
        
        // Update the pattern with care to synchronize changes
        if (threadIdx.x == 0) shared_pattern = shared_pattern << 1;
        __syncthreads(); // Ensure all threads see the updated pattern

        mask = mask << 1;
    }
    return;
}