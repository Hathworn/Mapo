#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    // Calculate the starting point for each block
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Early exit if pointer exceeds end pointer
    if (ptr >= (unsigned long*) end_ptr) return;

    // Efficiently set memory addresses using unrolling
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned long); i++) {
        ptr[i] = (unsigned long) &ptr[i];
    }
}