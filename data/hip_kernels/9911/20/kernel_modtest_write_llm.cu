#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    // Calculate the pointer offset for the current block
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Return if the calculated pointer is out of the bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Shared memory to reduce redundant memory accesses
    __shared__ unsigned int shared_ptr[BLOCKSIZE / sizeof(unsigned int)];

    // Load data into shared memory
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        shared_ptr[i] = ptr[i];
    }
    __syncthreads();

    // Apply first modification
    for (unsigned int i = threadIdx.x + offset; i < BLOCKSIZE / sizeof(unsigned int); i += MOD_SZ * blockDim.x) {
        shared_ptr[i] = p1;
    }
    __syncthreads();

    // Apply second modification
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        if (i % MOD_SZ != offset) {
            shared_ptr[i] = p2;
        }
    }
    __syncthreads();

    // Write the data back to the global memory
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        ptr[i] = shared_ptr[i];
    }
}