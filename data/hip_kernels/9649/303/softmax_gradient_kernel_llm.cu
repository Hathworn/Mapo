#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // Unroll loop to reduce instruction overhead
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Use warp reduction for better performance on reductions
    if (blockDim.x >= 32) {
        if (idx < 16) {
            reduction_buffer[idx] += reduction_buffer[idx + 16];
            reduction_buffer[idx] += reduction_buffer[idx + 8];
            reduction_buffer[idx] += reduction_buffer[idx + 4];
            reduction_buffer[idx] += reduction_buffer[idx + 2];
            reduction_buffer[idx] += reduction_buffer[idx + 1];
        }
    }

    if (idx == 0) {
        float sum = 0.0f;
        for (int i = 0; i < 32; ++i) {  // assuming SOFTMAX_NUM_THREADS is a multiple of 32
            sum += reduction_buffer[i];
        }
        reduction_buffer[0] = sum;
    }
    __syncthreads();

    // Compute gradient with updated tmp value
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}