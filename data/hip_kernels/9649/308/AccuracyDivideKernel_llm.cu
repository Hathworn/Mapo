#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Check if pointer is valid and avoid potential division errors
    if (accuracy != nullptr && N != 0) {
        *accuracy /= N;
    }
}