#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel with optimized memory access
__global__ void kernel_End(int *g_stochastic, int *g_count_blocks, int *g_counter) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = blockDim.x * gridDim.x;

    for (int idx = thid; idx < *g_counter; idx += totalThreads) { // Stride loop for coalesced memory access
        if (g_stochastic[idx] == 1) {
            atomicAdd(g_count_blocks, 1); // Perform atomic addition
        }
    }
}