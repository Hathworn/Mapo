#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel by applying thread coarsening and removing redundant calculations
__global__ void kernel_push_stochastic2(int *g_terminate, int *g_relabel_mask, int *g_push_reser, int *s_push_reser, int *d_stochastic, int *g_block_num, int width1)
{
    int blockOffset = blockIdx.y * (*g_block_num) + blockIdx.x;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int thid = y * width1 + x;

    // Coalesced memory access and reduced redundant calculations
    int push_diff = s_push_reser[thid] - g_push_reser[thid];
    if (push_diff != 0)
    {
        g_terminate[blockOffset] = 1;
    }

    if (g_relabel_mask[thid] == 1)
    {
        d_stochastic[blockOffset] = 1;
    }
}