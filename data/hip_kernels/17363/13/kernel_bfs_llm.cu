#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_bfs(int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight, int *g_graph_height, bool *g_pixel_mask, int vertex_num, int width, int height, int vertex_num1, int width1, int height1, bool *g_over, int *g_counter)
{
// Calculate thread ID
int thid = blockIdx.x * blockDim.x + threadIdx.x;

// Condition to ensure threads are within bounds and pixel mask is true
if (thid < vertex_num && g_pixel_mask[thid])
{
    int col = thid % width1, row = thid / width1;

    // Avoid boundary edges for optimization
    if (col > 0 && col < width - 1 && row > 0 && row < height - 1)
    {
        int height_r = g_graph_height[thid + 1];
        int height_l = g_graph_height[thid - 1];
        int height_d = g_graph_height[thid + width1];
        int height_u = g_graph_height[thid - width1];

        // Combine conditions for better warp efficiency
        if ((height_l == (*g_counter) && g_right_weight[thid - 1] > 0) ||
            (height_d == (*g_counter) && g_up_weight[thid + width1] > 0) ||
            (height_r == (*g_counter) && g_left_weight[thid + 1] > 0) ||
            (height_u == (*g_counter) && g_down_weight[thid - width1] > 0))
        {
            g_graph_height[thid] = (*g_counter) + 1;
            g_pixel_mask[thid] = false;
            *g_over = true;
        }
    }
}
}