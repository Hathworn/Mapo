#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_push_stochastic1(int *g_push_reser, int *s_push_reser, int *g_count_blocks, bool *g_finish, int *g_block_num, int width1)
{
    // Compute global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int thid = y * width1 + x;

    // Copy data from global to shared memory
    s_push_reser[thid] = g_push_reser[thid];

    // Check and update finish flag by the first thread
    if (thid == 0)
    {
        if (*g_count_blocks == 0)
        {
            *g_finish = false;
        }
    }
}