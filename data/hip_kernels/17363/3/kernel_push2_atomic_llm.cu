#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_push2_atomic(int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight, int *g_sink_weight, int *g_push_reser, int *g_pull_left, int *g_pull_right, int *g_pull_down, int *g_pull_up, int *g_relabel_mask, int *g_graph_height, int *g_height_write, int graph_size, int width, int rows, int graph_size1, int width1, int rows1)
{
    int x1 = threadIdx.x;
    int y1 = threadIdx.y;
    int x = blockIdx.x * blockDim.x + x1;
    int y = blockIdx.y * blockDim.y + y1;
    int thid = y * width1 + x;

    __shared__ int height_fn[356];

    int temp_mult = (y1 + 1) * 34 + x1 + 1, temp_mult1 = y1 * 32 + x1;

    height_fn[temp_mult] = g_graph_height[thid];

    if (x < width1 - 1)
        height_fn[temp_mult + 1] = (g_graph_height[thid + 1]);
    if (x > 0)
        height_fn[temp_mult - 1] = (g_graph_height[thid - 1]);
    if (y < rows1 - 1)
        height_fn[temp_mult + 34] = (g_graph_height[thid + width1]);
    if (y > 0)
        height_fn[temp_mult - 34] = (g_graph_height[thid - width1]);

    __syncthreads();

    int flow_push = g_push_reser[thid];
    int min_flow_pushed = flow_push;

    if (thid < graph_size1 && g_relabel_mask[thid] == 1 && x < width - 1 && x > 0 && y < rows - 1 && y > 0)
    {
        int temp_weight = g_sink_weight[thid];
        if (temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == 1)
        {
            if (temp_weight < flow_push) min_flow_pushed = temp_weight;
            g_sink_weight[thid] = temp_weight - min_flow_pushed;
            atomicSub(&g_push_reser[thid], min_flow_pushed);
        }

        for (int dir = 0; dir < 4; dir++)
        {
            int *weight, *opp_weight, offset;
            switch (dir) {
                case 0: weight = g_left_weight; opp_weight = g_right_weight; offset = -1; break;
                case 1: weight = g_up_weight; opp_weight = g_down_weight; offset = -width1; break;
                case 2: weight = g_right_weight; opp_weight = g_left_weight; offset = 1; break;
                default: weight = g_down_weight; opp_weight = g_up_weight; offset = width1; break;
            }
            temp_weight = weight[thid];
            if (temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + offset] + 1)
            {
                if (temp_weight < flow_push) min_flow_pushed = temp_weight;
                atomicSub(&weight[thid], min_flow_pushed);
                atomicAdd(&opp_weight[thid + offset], min_flow_pushed);
                atomicSub(&g_push_reser[thid], min_flow_pushed);
                atomicAdd(&g_push_reser[thid + offset], min_flow_pushed);
            }
            else if (dir == 0) atomicSub(&g_pull_left[thid + offset], 1);
            else if (dir == 1) atomicSub(&g_pull_up[thid + offset], 1);
            else if (dir == 2) atomicSub(&g_pull_right[thid + offset], 1);
            else atomicSub(&g_pull_down[thid + offset], 1);
        }
    }

    __syncthreads();

    min_flow_pushed = g_left_weight[thid];

    if (flow_push <= 0 || (g_left_weight[thid] == 0 && g_right_weight[thid] == 0 && g_down_weight[thid] == 0 && g_up_weight[thid] == 0 && g_sink_weight[thid] == 0))
        g_relabel_mask[thid] = 2;
    else
    {
        if (flow_push > 0 && ((height_fn[temp_mult] == height_fn[temp_mult - 1] + 1 && g_left_weight[thid] > 0) || 
                              (height_fn[temp_mult] == height_fn[temp_mult + 1] + 1 && g_right_weight[thid] > 0) || 
                              (height_fn[temp_mult] == height_fn[temp_mult + 34] + 1 && g_down_weight[thid] > 0) || 
                              (height_fn[temp_mult] == height_fn[temp_mult - 34] + 1 && g_up_weight[thid] > 0) || 
                              (height_fn[temp_mult] == 1 && g_sink_weight[thid] > 0))) 
            g_relabel_mask[thid] = 1;
        else 
            g_relabel_mask[thid] = 0;
    }

    __syncthreads();

    if (thid < graph_size1 && g_relabel_mask[thid] == 1 && x < width - 1 && x > 0 && y < rows - 1 && y > 0)
    {
        int temp_weight = g_sink_weight[thid];
        if (temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == 1)
        {
            if (temp_weight < flow_push) min_flow_pushed = temp_weight;
            g_sink_weight[thid] = temp_weight - min_flow_pushed;
            atomicSub(&g_push_reser[thid], min_flow_pushed);
        }

        for (int dir = 0; dir < 4; dir++)
        {
            int *weight, *opp_weight, offset;
            switch (dir) {
                case 0: weight = g_left_weight; opp_weight = g_right_weight; offset = -1; break;
                case 1: weight = g_up_weight; opp_weight = g_down_weight; offset = -width1; break;
                case 2: weight = g_right_weight; opp_weight = g_left_weight; offset = 1; break;
                default: weight = g_down_weight; opp_weight = g_up_weight; offset = width1; break;
            }
            temp_weight = weight[thid];
            if (temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + offset] + 1)
            {
                if (temp_weight < flow_push) min_flow_pushed = temp_weight;
                atomicSub(&weight[thid], min_flow_pushed);
                atomicAdd(&opp_weight[thid + offset], min_flow_pushed);
                atomicSub(&g_push_reser[thid], min_flow_pushed);
                atomicAdd(&g_push_reser[thid + offset], min_flow_pushed);
            }
            else if (dir == 0) atomicSub(&g_pull_left[thid + offset], 1);
            else if (dir == 1) atomicSub(&g_pull_up[thid + offset], 1);
            else if (dir == 2) atomicSub(&g_pull_right[thid + offset], 1);
            else atomicSub(&g_pull_down[thid + offset], 1);
        }
    }
}