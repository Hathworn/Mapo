#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_bfs(int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight, int *g_graph_height, bool *g_pixel_mask, int vertex_num, int width, int height, int vertex_num1, int width1, int height1, bool *g_over, int *g_counter)
{
    // Calculate thread index
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < vertex_num && g_pixel_mask[thid])
    {
        int col = thid % width1, row = thid / width1;

        // Check for boundary conditions
        if (col < width - 1 && col > 0 && row < height - 1 && row > 0)
        {
            // Load neighboring heights
            int height_l = g_graph_height[thid - 1];
            int height_r = g_graph_height[thid + 1];
            int height_d = g_graph_height[thid + width1];
            int height_u = g_graph_height[thid - width1];

            int counter_val = *g_counter;

            // Check BFS condition and update graph height
            if ((height_l == counter_val && g_right_weight[thid - 1] > 0) || 
                (height_d == counter_val && g_up_weight[thid + width1] > 0) || 
                (height_r == counter_val && g_left_weight[thid + 1] > 0) || 
                (height_u == counter_val && g_down_weight[thid - width1] > 0))
            {
                g_graph_height[thid] = counter_val + 1;
                g_pixel_mask[thid] = false;
                *g_over = true;
            }
        }
    }
}