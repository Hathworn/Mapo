#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_push_stochastic2(int *g_terminate, int *g_relabel_mask, int *g_push_reser, int *s_push_reser, int *d_stochastic, int *g_block_num, int width1)
{
    // Calculate global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int thid = y * width1 + x;

    // Check if s_push_reser and g_push_reser differ; set terminate flag
    if (s_push_reser[thid] != g_push_reser[thid])
    {
        g_terminate[blockIdx.y * (*g_block_num) + blockIdx.x] = 1;
    }

    // Check relabel mask; set stochastic flag
    if (g_relabel_mask[thid] == 1)
    {
        d_stochastic[blockIdx.y * (*g_block_num) + blockIdx.x] = 1;
    }
}