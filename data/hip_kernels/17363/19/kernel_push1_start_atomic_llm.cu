#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_push1_start_atomic( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight, int *g_sink_weight, int *g_push_reser, int *g_relabel_mask, int *g_graph_height, int *g_height_write, int graph_size, int width, int rows, int graph_size1, int width1, int rows1, int *d_relabel, int *d_stochastic, int *d_counter, bool *d_finish)
{
    int x1 = threadIdx.x ;
    int y1 = threadIdx.y ;
    int x  = blockIdx.x * blockDim.x + threadIdx.x ;
    int y  = blockIdx.y * blockDim.y + threadIdx.y ;
    int thid = y * width1 + x ;

    __shared__ int height_fn[356];

    int temp_mult = (y1 + 1) * 34 + x1 + 1;
    height_fn[temp_mult] = g_graph_height[thid];

    // Load neighboring elements
    if (threadIdx.x == 31 && x < width1 - 1)
        height_fn[temp_mult + 1] = g_graph_height[thid + 1];
    if (threadIdx.x == 0 && x > 0)
        height_fn[temp_mult - 1] = g_graph_height[thid - 1];
    if (threadIdx.y == 7 && y < rows1 - 1)
        height_fn[temp_mult + 34] = g_graph_height[thid + width1];
    if (threadIdx.y == 0 && y > 0)
        height_fn[temp_mult - 34] = g_graph_height[thid - width1];

    __syncthreads();

    int flow_push = g_push_reser[thid];
    int min_flow_pushed = flow_push;

    if (thid < graph_size1 && g_relabel_mask[thid] == 1 && x < width - 1 && x > 0 && y < rows - 1 && y > 0)
    {
        int temp_weight = g_sink_weight[thid];

        if (temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == 1)
        {
            if (temp_weight < flow_push) 
                min_flow_pushed = temp_weight;
            g_sink_weight[thid] = temp_weight - min_flow_pushed;
            atomicSub(&g_push_reser[thid], min_flow_pushed);

            flow_push -= min_flow_pushed;
        }
    }

    __syncthreads();

    // Evaluate relabel condition
    bool is_relabel_needed = (flow_push > 0) && (
        ((height_fn[temp_mult] == height_fn[temp_mult - 1] + 1) && g_left_weight[thid] > 0) ||
        ((height_fn[temp_mult] == height_fn[temp_mult + 1] + 1) && g_right_weight[thid] > 0) ||
        ((height_fn[temp_mult] == height_fn[temp_mult + 34] + 1) && g_down_weight[thid] > 0) ||
        ((height_fn[temp_mult] == height_fn[temp_mult - 34] + 1) && g_up_weight[thid] > 0) ||
        (height_fn[temp_mult] == 1 && g_sink_weight[thid] > 0)
    );
    
    g_relabel_mask[thid] = is_relabel_needed ? 1 : 0;

    if (thid < graph_size1 && x < width - 1 && x > 0 && y < rows - 1 && y > 0)
    {
        if (g_sink_weight[thid] > 0)
        {
            g_height_write[thid] = 1;
        }
        else
        {
            int min_height = graph_size;
            if (g_left_weight[thid] > 0 && min_height > height_fn[temp_mult - 1])
                min_height = height_fn[temp_mult - 1];
            if (g_right_weight[thid] > 0 && min_height > height_fn[temp_mult + 1])
                min_height = height_fn[temp_mult + 1];
            if (g_down_weight[thid] > 0 && min_height > height_fn[temp_mult + 34])
                min_height = height_fn[temp_mult + 34];
            if (g_up_weight[thid] > 0 && min_height > height_fn[temp_mult - 34])
                min_height = height_fn[temp_mult - 34];
            g_height_write[thid] = min_height + 1;
        }
    }
}