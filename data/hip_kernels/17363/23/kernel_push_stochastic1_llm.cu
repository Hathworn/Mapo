#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_push_stochastic1(int *g_push_reser, int *s_push_reser, int *g_count_blocks, bool *g_finish, int *g_block_num, int width1) {
    // Calculate thread ID
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int thid = y * width1 + x;

    // Copy data from global to shared memory
    if (thid < width1 * gridDim.y * blockDim.y) {  // Avoid out-of-bounds memory access
        s_push_reser[thid] = g_push_reser[thid];
    }

    // First thread checks and updates the finish flag
    if (thid == 0) {
        if (atomicExch(g_count_blocks, *g_count_blocks) == 0) {  // Use atomicExch for thread-safe reading
            *g_finish = false;
        }
    }
}