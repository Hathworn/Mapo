#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_push_atomic2(int *g_terminate, int *g_push_reser, int *s_push_reser, int *g_block_num, int width1)
{
    // Calculate unique thread ID across the grid
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int thid = y * width1 + x;

    // Check the condition and set the g_terminate flag using atomic operations
    if (s_push_reser[thid] != g_push_reser[thid]) {
        atomicOr(&g_terminate[blockIdx.y * (*g_block_num) + blockIdx.x], 1);
    }
}