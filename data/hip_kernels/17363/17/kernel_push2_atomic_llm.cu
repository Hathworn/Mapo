#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_push2_atomic( int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight, int *g_sink_weight, int *g_push_reser, int *g_pull_left, int *g_pull_right, int *g_pull_down, int *g_pull_up, int *g_relabel_mask, int *g_graph_height, int *g_height_write, int graph_size, int width, int rows, int graph_size1, int width1, int rows1)
{
    int x1 = threadIdx.x ;
    int y1 = threadIdx.y ;
    int x  = blockIdx.x * blockDim.x + threadIdx.x ;
    int y  = blockIdx.y * blockDim.y + threadIdx.y ;
    int thid = y * width1 + x ;

    __shared__ int height_fn[356];

    int temp_mult = (y1+1) * 34 + x1 + 1, temp_mult1 = y1 * 32 + x1 ;

    height_fn[temp_mult] = g_graph_height[thid] ;

    // Load neighboring heights conditionally
    if(threadIdx.x == 31 && x < width1 - 1) height_fn[temp_mult + 1] = g_graph_height[thid + 1];
    if(threadIdx.x == 0 && x > 0) height_fn[temp_mult - 1] = g_graph_height[thid - 1];
    if(threadIdx.y == 7 && y < rows1 - 1) height_fn[temp_mult + 34] = g_graph_height[thid + width1];
    if(threadIdx.y == 0 && y > 0) height_fn[temp_mult - 34] = g_graph_height[thid - width1];

    __syncthreads();

    int flow_push = g_push_reser[thid], min_flow_pushed = flow_push;

    if( thid < graph_size1 && g_relabel_mask[thid] == 1 && x < width-1 && x > 0 && y < rows-1 && y > 0 )
    {
        int temp_weight = g_sink_weight[thid];
        min_flow_pushed = flow_push;

        // Process sink weights
        if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == 1)
        {
            if(temp_weight < flow_push) min_flow_pushed = temp_weight;
            atomicSub(&g_sink_weight[thid], min_flow_pushed);
            atomicSub(&g_push_reser[thid], min_flow_pushed);
        }

        flow_push = g_push_reser[thid];
        min_flow_pushed = flow_push;

        // Process left weights
        temp_weight = g_left_weight[thid];
        if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 1] + 1)
        {
            if(temp_weight < flow_push) min_flow_pushed = temp_weight;
            atomicSub(&g_left_weight[thid], min_flow_pushed);
            atomicAdd(&g_right_weight[thid-1], min_flow_pushed);
            atomicSub(&g_push_reser[thid], min_flow_pushed);
            atomicAdd(&g_push_reser[thid-1], min_flow_pushed);
        }
        else atomicSub(&g_pull_left[thid-1], 1);

        flow_push = g_push_reser[thid];
        min_flow_pushed = flow_push;

        // Process up weights
        temp_weight = g_up_weight[thid];
        if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 34] + 1)
        {
            if(temp_weight < flow_push) min_flow_pushed = temp_weight;
            atomicSub(&g_up_weight[thid], min_flow_pushed);
            atomicAdd(&g_down_weight[thid-width1], min_flow_pushed);
            atomicSub(&g_push_reser[thid], min_flow_pushed);
            atomicAdd(&g_push_reser[thid-width1], min_flow_pushed);
        }
        else atomicSub(&g_pull_up[thid-width1], 1);

        flow_push = g_push_reser[thid];
        min_flow_pushed = flow_push;

        // Process right weights
        temp_weight = g_right_weight[thid];
        if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 1] + 1)
        {
            if(temp_weight < flow_push) min_flow_pushed = temp_weight;
            atomicSub(&g_right_weight[thid], min_flow_pushed);
            atomicAdd(&g_left_weight[thid+1], min_flow_pushed);
            atomicSub(&g_push_reser[thid], min_flow_pushed);
            atomicAdd(&g_push_reser[thid+1], min_flow_pushed);
        }
        else atomicSub(&g_pull_right[thid+1], 1);

        flow_push = g_push_reser[thid];
        min_flow_pushed = flow_push;

        // Process down weights
        temp_weight = g_down_weight[thid];
        if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 34] + 1)
        {
            if(temp_weight < flow_push) min_flow_pushed = temp_weight;
            atomicSub(&g_down_weight[thid], min_flow_pushed);
            atomicAdd(&g_up_weight[thid+width1], min_flow_pushed);
            atomicSub(&g_push_reser[thid], min_flow_pushed);
            atomicAdd(&g_push_reser[thid+width1], min_flow_pushed);
        }
        else atomicSub(&g_pull_down[thid+width1], 1);
    }

    __syncthreads();

    min_flow_pushed = g_left_weight[thid];
    flow_push = g_push_reser[thid];

    // Update relabel status
    if(flow_push <= 0 || (g_left_weight[thid] == 0 && g_right_weight[thid] == 0 && g_down_weight[thid] == 0 && g_up_weight[thid] == 0 && g_sink_weight[thid] == 0))
        g_relabel_mask[thid] = 2;
    else
    {
        if(flow_push > 0 && (
            (height_fn[temp_mult] == height_fn[temp_mult-1] + 1 && g_left_weight[thid] > 0) ||
            (height_fn[temp_mult] == height_fn[temp_mult+1] + 1 && g_right_weight[thid] > 0) ||
            (height_fn[temp_mult] == height_fn[temp_mult+34] + 1 && g_down_weight[thid] > 0) ||
            (height_fn[temp_mult] == height_fn[temp_mult-34] + 1 && g_up_weight[thid] > 0) ||
            (height_fn[temp_mult] == 1 && g_sink_weight[thid] > 0)
        ))
            g_relabel_mask[thid] = 1;
        else
            g_relabel_mask[thid] = 0;
    }

    __syncthreads();

    if(thid < graph_size1 && g_relabel_mask[thid] == 1 && x < width-1 && x > 0 && y < rows-1 && y > 0)
    {
        int temp_weight = g_sink_weight[thid];
        min_flow_pushed = flow_push;

        // Repeat sink weight processing for eligible threads
        if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == 1)
        {
            if(temp_weight < flow_push) min_flow_pushed = temp_weight;
            atomicSub(&g_sink_weight[thid], min_flow_pushed);
            atomicSub(&g_push_reser[thid], min_flow_pushed);
        }

        flow_push = g_push_reser[thid];
        min_flow_pushed = flow_push;

        // Repeat left weight processing for eligible threads
        temp_weight = g_left_weight[thid];
        if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 1] + 1)
        {
            if(temp_weight < flow_push) min_flow_pushed = temp_weight;
            atomicSub(&g_left_weight[thid], min_flow_pushed);
            atomicAdd(&g_right_weight[thid-1], min_flow_pushed);
            atomicSub(&g_push_reser[thid], min_flow_pushed);
            atomicAdd(&g_push_reser[thid-1], min_flow_pushed);
        }
        else atomicSub(&g_pull_left[thid-1], 1);

        flow_push = g_push_reser[thid];
        min_flow_pushed = flow_push;

        // Repeat up weight processing for eligible threads
        temp_weight = g_up_weight[thid];
        if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult - 34] + 1)
        {
            if(temp_weight < flow_push) min_flow_pushed = temp_weight;
            atomicSub(&g_up_weight[thid], min_flow_pushed);
            atomicAdd(&g_down_weight[thid-width1], min_flow_pushed);
            atomicSub(&g_push_reser[thid], min_flow_pushed);
            atomicAdd(&g_push_reser[thid-width1], min_flow_pushed);
        }
        else atomicSub(&g_pull_up[thid-width1], 1);

        flow_push = g_push_reser[thid];
        min_flow_pushed = flow_push;

        // Repeat right weight processing for eligible threads
        temp_weight = g_right_weight[thid];
        if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 1] + 1)
        {
            if(temp_weight < flow_push) min_flow_pushed = temp_weight;
            atomicSub(&g_right_weight[thid], min_flow_pushed);
            atomicAdd(&g_left_weight[thid+1], min_flow_pushed);
            atomicSub(&g_push_reser[thid], min_flow_pushed);
            atomicAdd(&g_push_reser[thid+1], min_flow_pushed);
        }
        else atomicSub(&g_pull_right[thid+1], 1);

        flow_push = g_push_reser[thid];
        min_flow_pushed = flow_push;

        // Repeat down weight processing for eligible threads
        temp_weight = g_down_weight[thid];
        if(temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == height_fn[temp_mult + 34] + 1)
        {
            if(temp_weight < flow_push) min_flow_pushed = temp_weight;
            atomicSub(&g_down_weight[thid], min_flow_pushed);
            atomicAdd(&g_up_weight[thid+width1], min_flow_pushed);
            atomicSub(&g_push_reser[thid], min_flow_pushed);
            atomicAdd(&g_push_reser[thid+width1], min_flow_pushed);
        }
        else atomicSub(&g_pull_down[thid+width1], 1);
    }
}
```
