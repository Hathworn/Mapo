#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_End(int *g_stochastic, int *g_count_blocks, int *g_counter)
{
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    int counter_val = *g_counter;
    
    // Ensure thread reads counter value only once
    if (thid < counter_val)
    {
        // Use predicate to eliminate unnecessary atomic operations
        int val = (g_stochastic[thid] == 1);
        if (val)
        {
            atomicAdd(g_count_blocks, 1);
        }
    }
}
```
