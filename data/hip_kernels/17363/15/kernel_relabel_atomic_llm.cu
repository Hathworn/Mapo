#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_relabel_atomic(int *g_left_weight, int *g_right_weight, int *g_down_weight, int *g_up_weight, int *g_sink_weight, int *g_push_reser, int *g_pull_left, int *g_pull_right, int *g_pull_down, int *g_pull_up, int *g_relabel_mask, int *g_graph_height, int *g_height_write, int graph_size, int width, int rows, int graph_size1, int width1, int rows1) {
    int x1 = threadIdx.x;
    int y1 = threadIdx.y;
    int x = blockIdx.x * blockDim.x + x1;
    int y = blockIdx.y * blockDim.y + y1;
    int thid = y * width1 + x;

    __shared__ int height_fn[356];
    
    int temp_mult = (y1 + 1) * 34 + x1 + 1;
    height_fn[temp_mult] = g_graph_height[thid];
    
    // Load border heights into shared memory for the halo region
    if (threadIdx.x == 31 && x < width1 - 1) height_fn[temp_mult + 1] = g_graph_height[thid + 1];
    if (threadIdx.x == 0 && x > 0) height_fn[temp_mult - 1] = g_graph_height[thid - 1];
    if (threadIdx.y == 7 && y < rows1 - 1) height_fn[temp_mult + 34] = g_graph_height[thid + width1];
    if (threadIdx.y == 0 && y > 0) height_fn[temp_mult - 34] = g_graph_height[thid - width1];
    
    __syncthreads();

    int flow_push = g_push_reser[thid];
    int current_height = height_fn[temp_mult];

    // Compute relabel mask
    if (flow_push <= 0 || (g_left_weight[thid] == 0 && g_right_weight[thid] == 0 && g_down_weight[thid] == 0 && g_up_weight[thid] == 0 && g_sink_weight[thid] == 0)) {
        g_relabel_mask[thid] = 2;
    } else {
        bool condition = (current_height == height_fn[temp_mult - 1] + 1 && g_left_weight[thid] > 0) ||
                         (current_height == height_fn[temp_mult + 1] + 1 && g_right_weight[thid] > 0) ||
                         (current_height == height_fn[temp_mult + 34] + 1 && g_down_weight[thid] > 0) ||
                         (current_height == height_fn[temp_mult - 34] + 1 && g_up_weight[thid] > 0) ||
                         (current_height == 1 && g_sink_weight[thid] > 0);
        g_relabel_mask[thid] = condition ? 1 : 0;
    }

    __syncthreads();

    // Update height if within valid region
    if (thid < graph_size1 && x < width - 1 && x > 0 && y < rows - 1 && y > 0) {
        if (g_sink_weight[thid] > 0) {
            g_height_write[thid] = 1;
        } else {
            int min_height = graph_size;
            int h_left = height_fn[temp_mult - 1];
            int h_right = height_fn[temp_mult + 1];
            int h_down = height_fn[temp_mult + 34];
            int h_up = height_fn[temp_mult - 34];

            if (g_left_weight[thid] > 0) min_height = min(min_height, h_left);
            if (g_right_weight[thid] > 0) min_height = min(min_height, h_right);
            if (g_down_weight[thid] > 0) min_height = min(min_height, h_down);
            if (g_up_weight[thid] > 0) min_height = min(min_height, h_up);

            g_height_write[thid] = min_height + 1;
        }
    }
}