#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_push_atomic2(int *g_terminate, int *g_push_reser, int *s_push_reser, int *g_block_num, int width1)
{
    // Improve thread indexing by using grid-stride loops
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int blocks_in_grid = gridDim.x * gridDim.y;

    for (int i = index; i < width1 * blocks_in_grid; i += stride)
    {
        if (s_push_reser[i] - g_push_reser[i] != 0)
        {
            // Calculate 2D block position and set termination flag
            int bx = i % width1;
            int by = (i / width1) / width1;
            g_terminate[by * (*g_block_num) + bx] = 1;
        }
    }
}