#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_bfs_t(int *g_push_reser, int *g_sink_weight, int *g_graph_height, bool *g_pixel_mask, int vertex_num, int width, int height, int vertex_num1, int width1, int height1)
{
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index is within range and pixel is not masked
    if (thid < vertex_num && g_pixel_mask[thid])
    {
        int col = thid % width1;
        int row = thid / width1;

        // Check conditions for setting height or unmasking pixels
        if (col > 0 && row > 0 && col < width - 1 && row < height - 1 && g_push_reser[thid] > 0)
        {
            g_graph_height[thid] = 1;
            g_pixel_mask[thid] = false;
        }
        else if (g_sink_weight[thid] > 0)
        {
            g_graph_height[thid] = -1;
            g_pixel_mask[thid] = false;
        }
    }
}