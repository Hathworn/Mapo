#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_push1_start_stochastic(int* g_left_weight, int* g_right_weight, int* g_down_weight, int* g_up_weight, int* g_sink_weight, int* g_push_reser, int* g_relabel_mask, int* g_graph_height, int* g_height_write, int graph_size, int width, int rows, int graph_size1, int width1, int rows1, int* d_relabel, int* d_stochastic, int* d_counter, bool* d_finish)
{
    // Calculate thread's block and grid position
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int thid = y * width1 + x;

    // Define shared memory array
    __shared__ int height_fn[356];

    // Calculate local thread position and indices
    int x1 = threadIdx.x;
    int y1 = threadIdx.y;
    int temp_mult = (y1 + 1) * 34 + x1 + 1;
    
    // Read height values into shared memory with boundary condition checks
    if (x < width1 && y < rows1)
        height_fn[temp_mult] = g_graph_height[thid];
    if (threadIdx.x == 31 && x < width1 - 1) 
        height_fn[temp_mult + 1] = g_graph_height[thid + 1];
    if (threadIdx.x == 0 && x > 0) 
        height_fn[temp_mult - 1] = g_graph_height[thid - 1];
    if (threadIdx.y == 7 && y < rows1 - 1) 
        height_fn[temp_mult + 34] = g_graph_height[thid + width1];
    if (threadIdx.y == 0 && y > 0) 
        height_fn[temp_mult - 34] = g_graph_height[thid - width1];

    __syncthreads();

    int flow_push = g_push_reser[thid];

    // Perform operations if within graph bounds
    if (thid < graph_size1 && g_relabel_mask[thid] == 1 && x < width - 1 && x > 0 && y < rows - 1 && y > 0) {
        int temp_weight = g_sink_weight[thid];
        int min_flow_pushed = flow_push;

        if (temp_weight > 0 && flow_push > 0 && height_fn[temp_mult] == 1) {
            min_flow_pushed = min(temp_weight, flow_push);
            atomicSub(&g_push_reser[thid], min_flow_pushed);
            g_sink_weight[thid] = temp_weight - min_flow_pushed;
            flow_push -= min_flow_pushed;
        }
    }

    __syncthreads();

    // Update relabel mask based on flow conditions
    bool condition = (flow_push > 0 &&
                      ((height_fn[temp_mult] == height_fn[temp_mult - 1] + 1 && g_left_weight[thid] > 0) ||
                       (height_fn[temp_mult] == height_fn[temp_mult + 1] + 1 && g_right_weight[thid] > 0) ||
                       (height_fn[temp_mult] == height_fn[temp_mult + 34] + 1 && g_down_weight[thid] > 0) ||
                       (height_fn[temp_mult] == height_fn[temp_mult - 34] + 1 && g_up_weight[thid] > 0) ||
                       (height_fn[temp_mult] == 1 && g_sink_weight[thid] > 0)));
    g_relabel_mask[thid] = condition ? 1 : 0;

    // Adjust height_write based on updated weights
    if (thid < graph_size1 && x < width - 1 && x > 0 && y < rows - 1 && y > 0) {
        if (g_sink_weight[thid] > 0) {
            g_height_write[thid] = 1;
        } else {
            int min_height = graph_size;
            int neighbor_weights[4] = { height_fn[temp_mult - 1], height_fn[temp_mult + 1], height_fn[temp_mult + 34], height_fn[temp_mult - 34] };
            int neighbor_graph_weights[4] = { g_left_weight[thid], g_right_weight[thid], g_down_weight[thid], g_up_weight[thid] };

            for (int i = 0; i < 4; ++i) {
                if (neighbor_graph_weights[i] > 0 && min_height > neighbor_weights[i]) {
                    min_height = neighbor_weights[i];
                }
            }
            g_height_write[thid] = min_height + 1;
        }
    }
}