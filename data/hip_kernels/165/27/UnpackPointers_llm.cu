#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    __shared__ unsigned int shared_minmax[16]; // Use shared memory for better performance
    int numPts = 0;

    for (int y = 0; y < h / 32; y++) {
        for (int x = 0; x < w; x += 16) {
            if (tx < 16) {
                shared_minmax[tx] = minmax[y * w + x + tx];
            }
            __syncthreads(); // Ensure all threads have loaded the value

            unsigned int val = shared_minmax[tx];
            if (val) {
                for (int k = 0; k < 32 && numPts < maxPts; k++) { // Check numPts condition in the loop condition
                    if (val & 0x1) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
            __syncthreads(); // Synchronize before next iteration
        }
    }
}