#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Use shared memory for coalesced access and reduced global memory access
    __shared__ float shared_Data[SUBTRACTM_H][SUBTRACTM_W];

    const int x = blockIdx.x * SUBTRACTM_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACTM_H + threadIdx.y;
    int sz = height * pitch;
    int p = threadIdx.z * sz + y * pitch + x;

    if (x < width && y < height) {
        // Load data into shared memory
        shared_Data[threadIdx.y][threadIdx.x] = d_Data[p];
        __syncthreads();

        // Perform computation using data in shared memory
        float value = shared_Data[threadIdx.y][threadIdx.x] - d_Data[p + sz];

        // Write the result back to global memory
        d_Result[p] = value;
    }
}