#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    const int p = y * width + x;

    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
        for (int ty = 1; ty < 16; ty++) {
            val = d_Data[p + ty * width];
            minvals[tx] = fminf(val, minvals[tx]); // Use fminf for min comparison
            maxvals[tx] = fmaxf(val, maxvals[tx]); // Use fmaxf for max comparison
        }
    } else {
        minvals[tx] = maxvals[tx] = d_Data[p - x];
    }

    __syncthreads();

    // Use warp shuffle to perform reduction within a warp
    for (int d = b / 2; d > 0; d >>= 1) {
        if (tx < d) {
            minvals[tx] = fminf(minvals[tx], minvals[tx + d]);
            maxvals[tx] = fmaxf(maxvals[tx], maxvals[tx + d]);
        }
        __syncthreads();
    }

    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}