#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    const int strideX = blockDim.x;
    const int strideY = gridDim.x; // Optimization: Use grids and blocks 

    // Optimization: Determine global thread index
    int globalX = blockIdx.x * strideX + tx;
    int globalY = blockIdx.y * strideY;
    int numPts = 0;

    for (int y = globalY; y < h; y += strideY * 32) // Optimization: Adjust loop range
    {
        for (int x = globalX; x < w; x += strideX) // Optimization: Adjust loop range
        {
            unsigned int val = minmax[y * w + x];
            
            if (val)
            {
                for (int k = 0; k < 32; k++)
                {
                    if (val & 0x1 && numPts < maxPts)
                    {
                        ptrs[16 * numPts + tx] = (y + k) * w + x;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}