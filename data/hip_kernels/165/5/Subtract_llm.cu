#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    // Calculate global thread index
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Calculate the 1D index in the linear memory
    int p = y * pitch + x;
    
    // Perform subtraction only for valid indices
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // No need for __syncthreads() here as threads within a block do not depend on each other
}