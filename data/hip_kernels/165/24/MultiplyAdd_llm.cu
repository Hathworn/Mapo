#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Calculate unique thread index for 2D blocks
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;

    // Check bounds condition
    if (x < width && y < height)
    {
        // Perform multiply-add operation
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
}