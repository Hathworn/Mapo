#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    // Use threadIdx and blockDim for better parallelism
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}