#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_renorm(float *data, const float value, const long size, const float maxnorm)
{
    __shared__ float buffer[32];
    long tx = threadIdx.x;
    long bx = blockIdx.x;
    long step = blockDim.x;
    float *row = data + size * bx;

    buffer[tx] = 0;

    // Calculate norm (each thread processes part of the row)
    for (long i = tx; i < size; i += step)
    {
        buffer[tx] += powf(fabsf(row[i]), value);
    }

    // Reduce using shared memory (optimize reduction pattern)
    __syncthreads();
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
    {
        if (tx < stride)
        {
            buffer[tx] += buffer[tx + stride];
        }
        __syncthreads();
    }

    // Clip norms and renormalize if necessary (only first thread needs to do it)
    if (tx == 0)
    {
        float norm = powf(buffer[0], 1 / value);
        if (norm > maxnorm)
        {
            norm = maxnorm / (norm + 1e-7f);
            buffer[0] = norm;  // Store norm for reuse
        }
        else
        {
            buffer[0] = 1.0f;  // No renormalization needed
        }
    }
    __syncthreads();

    // Apply renormalization
    for (long i = tx; i < size; i += step)
    {
        row[i] *= buffer[0];
    }
}