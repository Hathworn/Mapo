#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;

        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        // No probability mass or precision problems; just return the
        // first element
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithoutReplacement(hiprandStateMtgp32* state, int totalSamples, int sample, float* dest, long distributions, int categories, float* origDist, float* normDistPrefixSum) {
    // Optimize loop bounds calculation for fewer instructions
    long curDistBase = blockIdx.x * blockDim.y;

    while (curDistBase < distributions) {
        long curDist = curDistBase + threadIdx.y;

        // Use a single atomic operation to set the random value
        float r = hiprand_uniform(&state[blockIdx.x]);

        if (threadIdx.x == 0 && curDist < distributions) {
            int choice = binarySearchForMultinomial(
                normDistPrefixSum + curDist * categories,
                categories,
                r);
                
            dest[curDist * totalSamples + sample] = (float) choice + 1.0f;
            origDist[curDist * categories + choice] = 0.0f;
        }
        // Increment curDistBase by the number of threads in y dimension
        curDistBase += gridDim.x * blockDim.y;
    }
}