#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;

        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        start = 0; // Return the first element if no probability mass
    }

    return start;
}

__global__ void sampleMultinomialWithReplacement(hiprandStateMtgp32* state, int totalSamples, float* dest, long distributions, int categories, float* normDistPrefixSum) {
    long curDist = blockIdx.x; // Each block handles one distribution

    if (curDist < distributions) { // Ensure curDist is in range
        for (int sampleBase = 0; sampleBase < totalSamples; sampleBase += blockDim.y) {
            int sample = sampleBase + threadIdx.y; // Each thread calculates one sample

            float r = hiprand_uniform(&state[blockIdx.x]); // Generate random number

            if (threadIdx.x == 0 && sample < totalSamples) {
                // Find the bucket that a uniform sample lies in
                int choice = binarySearchForMultinomial(normDistPrefixSum + curDist * categories, categories, r);
                dest[curDist * totalSamples + sample] = (float)choice + 1.0f; // Store 1-based result
            }
        }
    }
}