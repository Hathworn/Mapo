#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate global thread index with block-wise distribution
    const uint tidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize each thread with a unique sequence number for random numbers
    hiprand_init(seed, tidx, 0, &state[tidx]);
}