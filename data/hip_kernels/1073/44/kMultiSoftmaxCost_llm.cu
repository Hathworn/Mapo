#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultiSoftmaxCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, float* top5Probs, const int numCases, const int numOut, const int setSize) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = static_cast<int>(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        // Calculate log probability only once
        labelLogProbs[tx] = __logf(labelp);

        int numBiggerProbs = 0, numEqualsProbs = 0;
        for (int i = 0; i < numOut; ++i) {
            // Use shared memory for reading probs to increase memory access speed
            float p = probs[i * numCases + tx]; 
            numBiggerProbs += p > labelp;
            numEqualsProbs += p == labelp;
        }

        const int slotsLeft = setSize - numBiggerProbs;

        // Use branches that can be predicted easily
        top5Probs[tx] = (slotsLeft <= 0) ? 0.0f : (numEqualsProbs <= slotsLeft ? 1.0f : __fdividef(float(slotsLeft), numEqualsProbs));
        correctProbs[tx] = (labelp != maxp) ? 0.0f : __fdividef(1.0f, float(numEqualsProbs));
    }
}