#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalThreads = blockDim.x * gridDim.x;

    // Use shared memory for performance improvement
    __shared__ float sharedSrc[blockDim.x];
    
    for (uint i = idx; i < tgtWidth * tgtHeight; i += totalThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Load data into shared memory using cooperative threads
        if (threadIdx.x < srcWidth) {
            sharedSrc[threadIdx.x] = src[srcY * srcWidth + srcX];
        }
        __syncthreads();  // Synchronize threads before proceeding
        
        // Access data from shared memory
        tgt[i] = sharedSrc[srcX];
    }
}