#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Use shared memory for counting max probability labels to reduce global memory access
        __shared__ int sharedMax[LOGREG_ERR_THREADS_X];
        if (threadIdx.x < numOut) {
            atomicAdd(&sharedMax[threadIdx.x], probs[threadIdx.x * numCases + tx] == maxp);
        }
        __syncthreads();

        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            for (int i = 0; i < LOGREG_ERR_THREADS_X; i++) {
                numMax += sharedMax[i];
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}