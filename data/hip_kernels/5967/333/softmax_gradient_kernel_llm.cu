#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define WARP_SIZE 32

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float shared_tmp[WARP_SIZE]; // Shared memory with Warp size

    float tmp = 0.0f;
    // Partial inner product in each thread
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }

    // Warp-level reduction for better shared memory efficiency
    __syncthreads();
    shared_tmp[idx] = tmp;
    if (idx < WARP_SIZE) {
        for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
            tmp += __shfl_down_sync(0xffffffff, tmp, offset);
        }
        if (idx == 0) {
            shared_tmp[0] = tmp; 
        }
    }
    __syncthreads();
   
    // Compute gradient
    tmp = shared_tmp[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}