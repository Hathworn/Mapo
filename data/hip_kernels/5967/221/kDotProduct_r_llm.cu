#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    // Calculate the global index for the current thread
    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;

    // Initialize shared memory for this thread
    shmem[threadIdx.x] = 0;

    // Unroll the loop to reduce the number of iterations, optimizing memory access
    for (uint offset = eidx; offset < numElements; offset += gridDim.x * DP_BLOCKSIZE) {
        shmem[threadIdx.x] += a[offset] * b[offset];
    }

    __syncthreads();

    // Optimize reduction within the block using warp shuffle operations
    #pragma unroll
    for (uint offset = warpSize/2; offset > 0; offset /= 2) {
        shmem[threadIdx.x] += __shfl_down_sync(0xffffffff, shmem[threadIdx.x], offset);
    }

    // Write the result of this block's reduction to the target array
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}