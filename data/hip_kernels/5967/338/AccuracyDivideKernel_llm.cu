#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load and store accuracy value using temporary variable to improve memory access efficiency
    float temp_accuracy = *accuracy;
    temp_accuracy /= N;
    *accuracy = temp_accuracy;
}