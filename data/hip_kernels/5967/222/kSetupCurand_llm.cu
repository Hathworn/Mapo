#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Initialize each thread's state with a unique seed and sequence number
    hiprand_init(seed, tidx, 0, &state[tidx]);
}