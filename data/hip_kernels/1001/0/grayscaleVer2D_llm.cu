#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grayscaleVer2D(uchar3* input, uchar3* output, int imageWidth, int imageHeight) {
    int tid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int tid_y = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Ensure within image bounds
    if (tid_x >= imageWidth || tid_y >= imageHeight) return;
    
    // Calculate 1D index from 2D indices
    int tid = tid_x + tid_y * imageWidth;
    
    // Efficiently compute grayscale value by avoiding division within the averaging process
    unsigned char grayValue = (input[tid].x + input[tid].y + input[tid].z) / 3;
    output[tid] = make_uchar3(grayValue, grayValue, grayValue); // Avoid repetition by leveraging symmetry
}