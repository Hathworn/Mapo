#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void testKernel(void) {
    // Use thread and block indices to reduce redundant calls
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) { // Only one thread needs to execute printf
        printf("Naber Millet!!\n");
    }
}