#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaussian_filter(unsigned *in, unsigned *out, int width, int height) {
    __shared__ unsigned int shared_data[4]; // Use shared memory to store inputs being processed

    int x = blockIdx.x * 2 + threadIdx.x;
    int y = blockIdx.y * 2 + threadIdx.y;

    if (x < width && y < height) {
        shared_data[threadIdx.y * 2 + threadIdx.x] = in[y * width + x]; // Load inputs into shared memory
    } else {
        shared_data[threadIdx.y * 2 + threadIdx.x] = 0; // Handle boundary conditions
    }

    __syncthreads();

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        unsigned int cikti = shared_data[0] + shared_data[1] + shared_data[2] + shared_data[3];

        out[blockIdx.y * width / 2 + blockIdx.x] = cikti; // Write result to output
    }
}