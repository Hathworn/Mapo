#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil_ld(unsigned *X, unsigned *out, int width, int height) {
    // Use shared memory for faster access
    __shared__ unsigned shared_X[34][34]; // extra padding to prevent out-of-bounds access
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x * blockDim.x;
    int by = blockIdx.y * blockDim.y;

    int x = bx + tx; 
    int y = by + ty;

    if (x >= width || y >= height) return; // boundary check

    // Load data into shared memory
    shared_X[ty + 1][tx + 1] = X[y * width + x];

    // Load boundary data for stencil operation
    if (tx == 0 && x > 0) shared_X[ty + 1][0] = X[y * width + x - 1];
    if (tx == 31 && x < width - 1) shared_X[ty + 1][33] = X[y * width + x + 1];
    if (ty == 0 && y > 0) shared_X[0][tx + 1] = X[(y - 1) * width + x];
    if (ty == 31 && y < height - 1) shared_X[33][tx + 1] = X[(y + 1) * width + x];

    __syncthreads();

    // Stencil computation
    int result = (kernel[0][0] * shared_X[ty][tx] +
                  kernel[0][1] * shared_X[ty][tx + 1] +
                  kernel[0][2] * shared_X[ty][tx + 2] +
                  kernel[1][0] * shared_X[ty + 1][tx] +
                  kernel[1][1] * shared_X[ty + 1][tx + 1] +
                  kernel[1][2] * shared_X[ty + 1][tx + 2] +
                  kernel[2][0] * shared_X[ty + 2][tx] +
                  kernel[2][1] * shared_X[ty + 2][tx + 1] +
                  kernel[2][2] * shared_X[ty + 2][tx + 2]);

    // Result clamping
    if (result < 0) out[y * width + x] = 0;
    else if (result > 255) out[y * width + x] = 255;
    else out[y * width + x] = result;
}