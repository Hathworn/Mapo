#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_ld(unsigned *in, unsigned *out){
    __shared__ int temp[BLOCK_SIZE + 2*RADIUS];
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x;

    // Load main block of data
    temp[lindex + RADIUS] = in[gindex];
    
    // Use only necessary threads to load halo data
    if(threadIdx.x < RADIUS){
        temp[lindex]                  = in[gindex - RADIUS];
        temp[lindex + BLOCK_SIZE + RADIUS] = in[gindex + BLOCK_SIZE];
    }
    
    __syncthreads();

    int result = 0;
    // Calculate result based on stencil
    for(int offset = -RADIUS; offset <= RADIUS; offset++){
        result += temp[lindex + offset + RADIUS];
    }

    // Store the result
    out[gindex] = result;
}