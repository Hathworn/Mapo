#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_dynamic_positions(float *arr, float t)
{
    int threadID = threadIdx.x;
    int blockID = blockIdx.x;
    int threads_per_block = blockDim.x;
    int i = blockID * threads_per_block + threadID;

    // Use a mask to reduce branching
    if (threadID < 3)
    {
        arr[i] = arr[i] * t;
    }
}