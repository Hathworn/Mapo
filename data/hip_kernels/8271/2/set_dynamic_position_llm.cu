#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_dynamic_position(float *arr, float t)
{
    int threadID = threadIdx.x;
    int blockID = blockIdx.x;
    int threads_per_block = blockDim.x;
    int i = blockID * threads_per_block + threadID;

    // Use shared memory to reduce global memory access
    __shared__ float sharedArr[3];
    if (threadID < 3)
    {
        sharedArr[threadID] = arr[i] * t;
        arr[i] = sharedArr[threadID];
    }
}