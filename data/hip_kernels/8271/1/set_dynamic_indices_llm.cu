#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_dynamic_indices(unsigned int *d_all_ib, unsigned int *d_ib, int flip)
{
    if (flip == blockIdx.x)
    {
        // Calculate global thread index once to avoid redundant calculations
        int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
        
        // Use a single read and write to reduce global memory accesses
        unsigned int value = d_all_ib[globalIdx];
        d_ib[threadIdx.x] = value;
        d_ib[3 + threadIdx.x] = 0;
    }
}