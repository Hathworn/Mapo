#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Simplified index calculation
    if (x < width && y < height)
    {
        int p = y * width + x; // Calculate linear index
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0]; // Perform operation
    }
    // __syncthreads() is not needed as there is no shared memory
}