#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;
    const int stride = blockDim.x; // Use block dimension for stride
    const int numRows = h / 32; // Precompute h/32 outside the loop

    for (int y = 0; y < numRows; ++y) {
        for (int x = tx; x < w; x += stride) { // Iterate with stride for x
            unsigned int val = minmax[y * w + x];
            if (val) {
                for (int k = 0; k < 32; ++k) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[stride * numPts + tx] = (y * 32 + k) * w + x;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}