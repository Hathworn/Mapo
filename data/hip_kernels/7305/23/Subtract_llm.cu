#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Use blockDim and blockIdx to calculate positions
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;
    
    // Remove unnecessary __syncthreads and use if condition efficiently
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}