#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Using more efficient shared memory usage and reducing repetitive computations
__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h) {
    __shared__ float data[16 * 15];
    __shared__ float hist[32];
    __shared__ float gauss[15];
    
    int tx = threadIdx.x;
    int bx = blockIdx.x;

    if (tx < 13) {
        hist[tx*32 + threadIdx.x] = 0.0f;
    }
    __syncthreads();

    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    if (tx < 15) {
        gauss[tx] = expf(i2sigma2 * (tx - 7) * (tx - 7));
    }

    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;
    int px = xp & 15;
    int x = tx - px;

    // Load data into shared memory
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = min(max(xp + x, 0), w - 1);
        int yi = min(max(yp + y, 0), h - 1);
        if (x >= 0 && x < 15) {
            data[memPos] = g_Data[yi * w + xi];
        }
    }
    __syncthreads();

    // Calculate gradients and populate histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = static_cast<int>(16.0f * atan2f(dy, dx) / 3.1416f + 16.5f) & 31;
            float grad = sqrtf(dx * dx + dy * dy);
            atomicAdd(&hist[bin], grad * gauss[x] * gauss[y]);
        }
    }
    __syncthreads();

    // Reduce histogram across threads
    if (tx < 16) {
        for (int i = 16; i > 0; i /= 2) {
            hist[tx] += hist[tx + i];
            __syncthreads();
        }
    }

    // Identify primary and secondary peaks in histogram
    if (tx == 0) {
        float maxval1 = 0.0f, maxval2 = 0.0f;
        int i1 = -1, i2 = -1;

        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }

        // Compute peak angle using quadratic interpolation
        float val1 = hist[(i1 + 1) & 31];
        float val2 = hist[(i1 + 31) & 31];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Orient[bx] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);

        if (maxval2 < 0.8f * maxval1) i2 = -1;

        if (i2 >= 0) {
            val1 = hist[(i2 + 1) & 31];
            val2 = hist[(i2 + 31) & 31];
            peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Orient[bx + maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        } else {
            d_Orient[bx + maxPts] = i2;
        }
    }
}