#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Simplified block width calculation
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Simplified block height calculation
    int p = y * pitch + x;
    
    if (x < width && y < height) // Efficient boundary check without syncthreads
    {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}
```
