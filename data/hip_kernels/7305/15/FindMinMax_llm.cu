#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    int p = y * pitch + x;
    float val;

    if (x < width) {
        val = d_Data[p];
    } else {
        val = d_Data[p - x];
    }
    
    minvals[tx] = val; // Initialize shared memory
    maxvals[tx] = val;
    __syncthreads();
    
    for (int ty = 1; ty < 16; ty++) {
        p += pitch;
        if (x < width) {
            val = d_Data[p];
            if (val < minvals[tx]) minvals[tx] = val; // Update for new row
            if (val > maxvals[tx]) maxvals[tx] = val;
        }
    }
    __syncthreads();

    // Unrolled reduction to improve efficiency
    int mod = 1;
    for (int d = b / 2; d > 0; d >>= 1) {
        if (tx < d) {
            if (minvals[tx + d] < minvals[tx]) minvals[tx] = minvals[tx + d];
            if (maxvals[tx + d] > maxvals[tx]) maxvals[tx] = maxvals[tx + d];
        }
        __syncthreads();
    }
    
    if (tx == 0) { // Write the result for this block
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}