#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    const int strideY = w * 32; // Pre-compute strideY for efficiency
    int numPts = 0;

    for (int y = 0; y < h / 32; y++) {
        for (int x = 0; x < w; x += 16) {
            unsigned int val = minmax[y * w + x + tx];

            if (val) {
                for (int k = 0; k < 32 && numPts < maxPts; k++) { // Limit loop by maxPts
                    if (val & 0x1) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}