#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Optimization: Use variables to avoid repeated multiplication
    const int x = blockIdx.x * SUBTRACTM_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACTM_H + threadIdx.y;
    int sz = height * pitch;
    int p = threadIdx.z * sz + y * pitch + x;

    // Check within bounds before accessing memory
    if (x < width && y < height) {
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
    // Remove unnecessary __syncthreads() since there are no shared memory operations or dependencies
}