#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;  // Simplified index computation
    const int y = blockIdx.y * 16;
    int p = y * width + x;

    // Initialize shared memory with first data point or zero if out of bounds
    float val = (x < width) ? d_Data[p] : 0.0f;
    minvals[tx] = val;
    maxvals[tx] = val;

    // Loop over y dimension
    for (int ty = 1; ty < 16; ty++) {
        p += width; 
        if (x < width && p / width < height) {  // Ensure we're within bounds
            val = d_Data[p]; 
            if (val < minvals[tx]) minvals[tx] = val;
            if (val > maxvals[tx]) maxvals[tx] = val;
        }
    }
    __syncthreads();

    // Reduction to find block min and max
    for (int d = b / 2; d > 0; d >>= 1) {
        if (tx < d && tx + d < b) {
            if (minvals[tx + d] < minvals[tx]) minvals[tx] = minvals[tx + d];
            if (maxvals[tx + d] > maxvals[tx]) maxvals[tx] = maxvals[tx + d];
        }
        __syncthreads();
    }

    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}