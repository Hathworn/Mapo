#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int add_d(int a, int b)
{
    // Remove redundant print statement for performance
    return a * b;
}

__global__ void add2(int a, int b, int *c)
{
    // Pre-compute add_d result without intermediate print
    int result = add_d(a, b);
    
    // Directly store result to global memory
    *c = result;
    
    // Remove unnecessary print statements to reduce kernel overhead
}