#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel function for better parallelism
__global__ void gSetSparse(float* out, const size_t* indices, const float* values, int length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < length) {
        out[indices[index]] = values[index];
    }
}