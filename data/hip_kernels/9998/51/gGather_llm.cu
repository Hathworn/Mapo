#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gGather(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < sparseSize) { // Combine conditions to reduce branching
        int effectiveIndex = sparseIndices[idx] + offset; // Calculate this once
        if(effectiveIndex >= 0 && effectiveIndex < denseSize) { // Simplified boundary check
            sparseData[idx] = denseData[effectiveIndex];
        }
    }
}