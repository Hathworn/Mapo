#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gMaxPoolingBackward(float* adj, int adjRows, int adjCols, float* in, float* adjIn, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
    // Calculate unique global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check for thread ID within bounds
    if(tid >= adjRows * adjCols)
        return;

    // Calculate row and column indices
    int rowId = tid / adjCols;
    int colId = tid % adjCols;

    // Determine the width for the last row
    int currentWidth = (colId == adjRows - 1) ? lastWidth : width;

    // Pointer to the start of the current row in the input
    float* b = in + (rowId * inCols) + (colId * width);

    // Pointer to the corresponding section of the mask
    float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;

    // Find the index of the max element considering the mask
    size_t currentMaxIdx = 0;
    for(int i = 1; i < currentWidth; ++i) {
        if(b[i] * localMask[i] > b[currentMaxIdx] * localMask[currentMaxIdx]) {
            currentMaxIdx = i;
        }
    }

    // Update adjIn with the adjusted gradient
    adjIn[(rowId * inCols) + (colId * width) + currentMaxIdx] += adj[rowId * adjCols + colId];
}