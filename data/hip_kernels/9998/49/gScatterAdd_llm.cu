#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gScatterAdd(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    // Use a single variable to calculate the global index and ensure correct range
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < sparseSize) {
        int denseIdx = sparseIndices[idx] + offset; // Calculate dense index once
        // Simplify conditional check and prevent out-of-bounds write
        if(denseIdx >= 0 && denseIdx < denseSize) {
            atomicAdd(&denseData[denseIdx], sparseData[idx]); // Use atomicAdd for thread safety
        }
    }
}