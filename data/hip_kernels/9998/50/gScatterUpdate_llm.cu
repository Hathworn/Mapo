#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gScatterUpdate(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    // Calculate global thread index for the denseData update
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure idx is within sparseSize bounds
    if(idx < sparseSize) {
        int targetIndex = sparseIndices[idx] + offset;

        // Optimize index range check using a simpler constraint
        if(targetIndex >= 0 && targetIndex < denseSize) {
            // Perform update only if targetIndex is valid
            denseData[targetIndex] = sparseData[idx];
        }
    }
}