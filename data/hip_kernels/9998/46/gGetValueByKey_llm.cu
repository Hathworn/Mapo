#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gGetValueByKey(float* d_in, float* d_out, int* indeces, int n) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    // Ensure all threads in warp access memory coalesced
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        int index = indeces[i];
        d_out[i] = d_in[index];
    }
}