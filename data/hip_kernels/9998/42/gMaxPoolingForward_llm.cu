#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gMaxPoolingForward(float* out, int outRows, int outCols, float* in, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if(tid >= outRows * outCols) return; // Boundary check

    int rowId = tid / outCols; // Correct rowId calculation
    int colId = tid % outCols; // Correct colId calculation

    // Pointer to current input block
    float* b = in + (rowId * inCols) + (colId * width); 
    // Pointer to relevant mask block
    float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;

    if(colId == outCols - 1) {
        width = lastWidth; // Adjust width for the last column
    }

    // Initialize max with first element
    float currentMax = b[0] * localMask[0];
    for(int i = 1; i < width; ++i) {
        float maskedValue = b[i] * localMask[i]; // Compute only once
        if(maskedValue > currentMax) {
            currentMax = maskedValue;
        }
    }

    // Store the result in the proper position
    out[rowId * outCols + colId] = currentMax;
}