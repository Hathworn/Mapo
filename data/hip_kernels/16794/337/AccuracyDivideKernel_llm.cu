#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
  // Use shared memory for faster access
  __shared__ float sharedAccuracy;
  if (threadIdx.x == 0) {
    sharedAccuracy = *accuracy;
  }
  __syncthreads();
  
  // Perform division using shared memory
  sharedAccuracy /= N;
  __syncthreads();
  
  if (threadIdx.x == 0) {
    *accuracy = sharedAccuracy;
  }
}