#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gradient_and_subtract_kernel(float * in, float * grad_x, float * grad_y, float * grad_z)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Use a single return to check bounds; saves unnecessary calculations
    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
        return;

    // Calculate base index once for reuse; avoids redundant calculations
    long int base_id = (k * c_Size.y + j) * c_Size.x + i;

    // Only compute required indices if within bounds
    if (i < c_Size.x - 1) {
        long int id_x = base_id + 1;
        grad_x[base_id] -= ((in[id_x] - in[base_id]) / c_Spacing.x);
    }
    if (j < c_Size.y - 1) {
        long int id_y = base_id + c_Size.x;
        grad_y[base_id] -= ((in[id_y] - in[base_id]) / c_Spacing.y);
    }
    if (k < c_Size.z - 1) {
        long int id_z = base_id + (c_Size.y * c_Size.x);
        grad_z[base_id] -= ((in[id_z] - in[base_id]) / c_Spacing.z);
    }
}