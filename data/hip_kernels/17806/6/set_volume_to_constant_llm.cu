#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_volume_to_constant(float * out, float value)
{
    // Calculate global linear thread index for improved performance.
    unsigned int id = blockIdx.z * blockDim.z * c_Size.y * c_Size.x +
                      blockIdx.y * blockDim.y * c_Size.x +
                      blockIdx.x * blockDim.x + 
                      threadIdx.z * c_Size.y * c_Size.x +
                      threadIdx.y * c_Size.x +
                      threadIdx.x;
    
    // Ensure we only write to valid indices.
    if (id >= c_Size.x * c_Size.y * c_Size.z)
        return;

    // Set the output to the constant value.
    out[id] = value;
}