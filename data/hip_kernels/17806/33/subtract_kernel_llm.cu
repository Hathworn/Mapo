#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subtract_kernel(float * in1, float * in2, float * out)
{
    // Calculate linear thread ID
    unsigned int id = blockIdx.z * blockDim.z * c_Size.y * c_Size.x +
                      blockIdx.y * blockDim.y * c_Size.x +
                      blockIdx.x * blockDim.x +
                      threadIdx.z * c_Size.y * c_Size.x +
                      threadIdx.y * c_Size.x +
                      threadIdx.x;

    // Check if id is within bounds
    if (id >= c_Size.x * c_Size.y * c_Size.z)
        return;

    // Perform subtraction
    out[id] = in1[id] - in2[id];
}