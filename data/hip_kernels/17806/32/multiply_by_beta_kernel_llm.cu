#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiply_by_beta_kernel(float *input, float *output, float beta)
{
    // Calculate the flattened global id for the 3D grid
    unsigned int id = (blockIdx.z * blockDim.z + threadIdx.z) * c_Size.y * c_Size.x +
                      (blockIdx.y * blockDim.y + threadIdx.y) * c_Size.x +
                      (blockIdx.x * blockDim.x + threadIdx.x);

    // Check if within bounds, leveraging short-circuit evaluation
    if (blockIdx.x * blockDim.x + threadIdx.x < c_Size.x &&
        blockIdx.y * blockDim.y + threadIdx.y < c_Size.y &&
        blockIdx.z * blockDim.z + threadIdx.z < c_Size.z)
    {
        output[id] = input[id] * beta;
    }
}
```
