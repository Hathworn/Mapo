#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void magnitude_threshold_kernel(float *grad_x, float *grad_y, float *grad_z, float gamma)
{
    // Use shared memory for the critical computations
    __shared__ float shared_grad_x[BLOCK_SIZE];
    __shared__ float shared_grad_y[BLOCK_SIZE];
    __shared__ float shared_grad_z[BLOCK_SIZE];

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
        return;

    long int id = (k * c_Size.y + j) * c_Size.x + i;

    // Load data into shared memory
    shared_grad_x[threadIdx.x] = grad_x[id];
    shared_grad_y[threadIdx.x] = grad_y[id];
    shared_grad_z[threadIdx.x] = grad_z[id];

    __syncthreads(); // Ensure all threads have loaded their data

    float norm = sqrt(shared_grad_x[threadIdx.x] * shared_grad_x[threadIdx.x] + 
                      shared_grad_y[threadIdx.x] * shared_grad_y[threadIdx.x] + 
                      shared_grad_z[threadIdx.x] * shared_grad_z[threadIdx.x]);

    if (norm > gamma)
    {
        float ratio = gamma / norm;
        shared_grad_x[threadIdx.x] *= ratio;
        shared_grad_y[threadIdx.x] *= ratio;
        shared_grad_z[threadIdx.x] *= ratio;

        // Write data back to global memory
        grad_x[id] = shared_grad_x[threadIdx.x];
        grad_y[id] = shared_grad_y[threadIdx.x];
        grad_z[id] = shared_grad_z[threadIdx.x];
    }
}