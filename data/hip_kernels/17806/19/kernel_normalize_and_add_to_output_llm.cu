#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_normalize_and_add_to_output(float *dev_vol_in, float *dev_vol_out, float *dev_accumulate_weights, float *dev_accumulate_values) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= c_volSize.x || j >= c_volSize.y || k >= c_volSize.z) {
        return;
    }

    // Calculate the index using row-major order
    long int out_idx = i + (j + k * c_volSize.y) * c_volSize.x;

    float eps = 1e-6;

    // Conditional normalization and addition
    if (c_normalize) {
        float weight = dev_accumulate_weights[out_idx];
        dev_vol_out[out_idx] = (fabs(weight) > eps) ? 
            (dev_vol_in[out_idx] + dev_accumulate_values[out_idx] / weight) :
            dev_vol_in[out_idx];
    } else {
        dev_vol_out[out_idx] = dev_vol_in[out_idx] + dev_accumulate_values[out_idx];
    }
}