#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use loop unrolling to improve performance
    int gridSize = blockDim.x * gridDim.x;
    while (idx < N) {
        a[idx] = a[idx] * a[idx];
        idx += gridSize;
    }
}