#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Compute unique linear thread index
    int index = threadIdx.x + blockIdx.y * blockDim.x;

    // Compute base pointers for gradInput/gradOutput/indices for the current batch
    float *gradInput_data = gradInput + blockIdx.x * input_w * input_n + index * input_n * dW;
    float *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + index * input_n;
    float *indices_data = indices + blockIdx.x * output_w * input_n + index * input_n;

    // Ensure thread index is within bounds
    if (index < output_w) {
        // For all features
        for (int feat = 0; feat < input_n; ++feat) {
            // Atomic add to ensure correctness with concurrent writes
            atomicAdd(&gradInput_data[(int)indices_data[feat] * input_n + feat], gradOutput_data[feat]);
        }
    }
}