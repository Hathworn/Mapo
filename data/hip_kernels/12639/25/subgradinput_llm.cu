#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinput(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Declare shared memory for weights to reduce global memory accesses
    __shared__ float shared_weight;

    // Load weight into shared memory, ensured by single thread per block
    if(threadIdx.x == 0 && threadIdx.y == 0) {
        shared_weight = weight[blockIdx.x % input_n];
    }
    __syncthreads(); // Synchronize to ensure shared memory is loaded

    // Compute output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute input/output offsets
    int o = blockIdx.x;
    int i = o;
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;

    // Iterate over output height
    for (int yy = blockDim.y * blockIdx.y + threadIdx.y; yy < output_h; yy += blockDim.y * gridDim.y) {
        // Iterate over output width
        for (int xx = threadIdx.x; xx < output_w; xx += blockDim.x) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            // Use shared weight
            float z = (*ptr_gradOutput) * shared_weight;
            // Directly iterate over kernel height and width, removing unnecessary pointers
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    ptr_gradInput[kx] += z;
                }
                ptr_gradInput += input_w;
            }
        }
    }
}