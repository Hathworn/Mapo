#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // iterators
    int xx, yy;
    
    // output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;
    
    // compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;
    
    int yy_start = blockIdx.y * blockDim.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;
    
    // select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    
    // get weight
    float the_weight = weight[k];
    
    // compute gradInput
    for (yy = yy_start; yy < output_h; yy += yy_step) {
        for (xx = xx_start; xx < output_w; xx += xx_step) {
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput * the_weight;
            float *ptr_gradInput = gradInput + (yy * dH) * input_w + xx * dW;
            
            // Unroll kH loop for coalesced memory access
            for (int ky = 0; ky < kH; ky++, ptr_gradInput += input_w) {
                // Unroll kW loop for improved parallel execution
                for (int kx = 0; kx < kW; kx++) {
                    atomicAdd(&ptr_gradInput[kx], z);
                }
            }
        }
    }
}