#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void atomicadaptivemaxgradinput( float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w )
{
    // compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // compute gradInput
    for(int yy = yy_start; yy < output_h; yy += yy_step) {

        int y_start = __float2int_rd(float(yy) / output_h * input_h); // Use faster rounding instruction

        for(int xx = xx_start; xx < output_w; xx += xx_step) {

            int x_start = __float2int_rd(float(xx) / output_w * input_w); // Use faster rounding instruction

            float *ptr_gradInput = gradInput + y_start * input_w + x_start;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float *ptr_ind_x = indices_x + yy * output_w + xx;
            float *ptr_ind_y = indices_y + yy * output_w + xx;
            float z = *ptr_gradOutput;

            int argmax_x = __float2int_rd(*ptr_ind_x) - 1; // Optimize with faster rounding and index calculation
            int argmax_y = __float2int_rd(*ptr_ind_y) - 1; // Optimize with faster rounding and index calculation

            // atomic add since different threads could update the same variable
            atomicAdd(&(ptr_gradInput[argmax_x + argmax_y * input_w]), z);
        }
    }
}