#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx = threadIdx.x + blockIdx.y * blockDim.x;
    int yy_start = blockDim.y * blockIdx.z + threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y * gridDim.z;
    
    float the_weight = weight[k];
    float the_bias = bias[k];
    
    output += o * output_w * output_h;
    input += i * input_w * input_h;

    if (xx < output_w) {
        for (int yy = yy_start; yy < yy_end; yy += yy_step) {
            float *ptr_input = input + yy * dH * input_w + xx * dW;
            float *ptr_output = output + yy * output_w + xx;
            float sum = 0;
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    sum += ptr_input[kx];
                }
                ptr_input += input_w; // next input line
            }
            *ptr_output = the_weight * sum + the_bias;
        }
    }
}