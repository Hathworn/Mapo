#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate time index and batch index
    int timeIdx = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    if (timeIdx >= output_w) return; // Early exit if out of bounds

    int batchIdx = blockIdx.x;
    int inputOffset = batchIdx * input_w * input_n + timeIdx * input_n * dW;
    int outputOffset = batchIdx * output_w * input_n + timeIdx * input_n;

    float *gradInput_data = gradInput + inputOffset;
    float *gradOutput_data = gradOutput + outputOffset;
    float *indices_data = indices + outputOffset;

    // Load gradOutput data and update gradInput
    for (int feat = 0; feat < input_n; ++feat) {
        int idx = (int)indices_data[feat];
        atomicAdd(&gradInput_data[idx * input_n + feat], gradOutput_data[feat]);  // Ensure atomic operation for potential race conditions
    }
}