#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_LookupTable_accGradParametersKernel( float *input,  float *indices, float *gradOutput, float *gradWeight, float *count, float defaultScale, long numel, long stride)
{
    int idx = blockIdx.x * 4 + threadIdx.y;

    // Calculate unique warp-level ID
    const int laneID = threadIdx.x % WARP_SIZE;

    // Number of values processed by each thread (grain size)
    const int SZ = 4;

    if (idx < numel && (idx == 0 || input[idx] != input[idx - 1]))
    {
        do
        {
            const int startFeature = laneID + blockIdx.y * blockDim.x * SZ; // Utilize laneID
            const int weightRow = ((int) input[idx] - 1) * stride;
            const int gradOutputRow = ((int) indices[idx] - 1) * stride;
            const float scale = count ? defaultScale / count[idx] : defaultScale;

            float gradient[SZ] = {0.0f}; // Initialize gradient array
            float weight[SZ] = {0.0f};   // Initialize weight array

            #pragma unroll
            for (int ii = 0; ii < SZ; ii++)
            {
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride)
                {
                    gradient[ii] = gradOutput[gradOutputRow + featureDim];
                    weight[ii] = gradWeight[weightRow + featureDim];
                }
            }

            #pragma unroll
            for (int ii = 0; ii < SZ; ii++)
            {
                weight[ii] += gradient[ii] * scale;
            }

            #pragma unroll
            for (int ii = 0; ii < SZ; ii++)
            {
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride)
                {
                    gradWeight[weightRow + featureDim] = weight[ii];
                }
            }

            // Advance index
            idx += WARP_SIZE; // Advance by the warp size

        } while (idx < numel && input[idx] == input[idx - 1]);
    }
}