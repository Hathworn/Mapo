#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "getForces_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float4 *g_pos = NULL;
hipMalloc(&g_pos, XSIZE*YSIZE);
float3 *force = NULL;
hipMalloc(&force, XSIZE*YSIZE);
int offset = 2;
int device_ct = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
getForces<<<gridBlock, threadBlock>>>(g_pos,force,offset,device_ct);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
getForces<<<gridBlock, threadBlock>>>(g_pos,force,offset,device_ct);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
getForces<<<gridBlock, threadBlock>>>(g_pos,force,offset,device_ct);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}