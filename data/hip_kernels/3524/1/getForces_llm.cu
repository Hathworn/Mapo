#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NR_NEUTRONS 8
#define NR_ELECTRONS 8
#define NR_PROTONS 8

#define MASS_PROTON 1.007276
#define MASS_NEUTRON 1.008664
#define MASS_ELECTRON 5.485799e-4

#define BLOCK 256

#define XWindowSize 2500
#define YWindowSize 2500

#define DRAW 10
#define DAMP 1.0

#define DT 0.001
#define STOP_TIME 10.0

#define G 6.67408E-11
#define H 1.0

#define EYE 8.5
#define FAR 80.0

#define SHAPE_CT 24
#define SHAPE_SIZE 256
#define PATH "./objects/Tokamak_256.obj"
#define N 16*16*16

float4 *p;
float3 *v, *f, *reactor,*r_GPU0, *r_GPU1;
float4 *p_GPU0, *p_GPU1;

__device__ float3 getBodyBodyForce(float4 p0, float4 p1){
    float3 f;
    float dx = p1.x - p0.x;
    float dy = p1.y - p0.y;
    float dz = p1.z - p0.z;
    float r2 = dx*dx + dy*dy + dz*dz;
    float inv_r = 1/sqrt(r2);

    float force  = (G*p0.w*p1.w)/(r2);

    f.x = force*dx*inv_r;
    f.y = force*dy*inv_r;
    f.z = force*dz*inv_r;

    return(f);
}

__global__ void getForces(float4 *g_pos, float3 *force, int offset, int device_ct){
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    float3 forceSum = make_float3(0.0f, 0.0f, 0.0f);
    float4 posMe = g_pos[id + offset];

    __shared__ float4 shPos[BLOCK];

    // Iterate through grids and calculate forces
    for (int j = 0; j < gridDim.x * device_ct; j++) {
        // Load positions to shared memory
        if (threadIdx.x < BLOCK) {
            shPos[threadIdx.x] = g_pos[threadIdx.x + blockDim.x * j];
        }
        __syncthreads();

        // Unroll the loop to improve performance
        #pragma unroll 32
        for (int i = 0; i < blockDim.x; i++) {
            int ii = i + blockDim.x * j;
            if (ii != id + offset && ii < N) {
                float3 force_b2b = getBodyBodyForce(posMe, shPos[i]);
                forceSum.x += force_b2b.x;
                forceSum.y += force_b2b.y;
                forceSum.z += force_b2b.z;
            }
        }
        __syncthreads();
    }

    // Write the computed forces for each particle
    if (id < N){
        force[id] = forceSum;
    }
}