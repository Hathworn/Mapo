#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "getForcesMag_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float4 *g_pos = NULL;
hipMalloc(&g_pos, XSIZE*YSIZE);
float3 *vel = NULL;
hipMalloc(&vel, XSIZE*YSIZE);
float3 *force = NULL;
hipMalloc(&force, XSIZE*YSIZE);
int offset = 2;
float3 *g_reactor = NULL;
hipMalloc(&g_reactor, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
getForcesMag<<<gridBlock, threadBlock>>>(g_pos,vel,force,offset,g_reactor);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
getForcesMag<<<gridBlock, threadBlock>>>(g_pos,vel,force,offset,g_reactor);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
getForcesMag<<<gridBlock, threadBlock>>>(g_pos,vel,force,offset,g_reactor);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}