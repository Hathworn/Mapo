#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NR_NEUTRONS 8
#define NR_ELECTRONS 8
#define NR_PROTONS 8

#define MASS_PROTON 1.007276
#define MASS_NEUTRON 1.008664
#define MASS_ELECTRON 5.485799e-4

#define BLOCK 256

#define XWindowSize 2500
#define YWindowSize 2500

#define DRAW 10
#define DAMP 1.0

#define DT 0.001
#define STOP_TIME 10.0

#define G 6.67408E-11
#define H 1.0

#define EYE 8.5
#define FAR 80.0

#define SHAPE_CT 24
#define SHAPE_SIZE 256
#define PATH "./objects/Tokamak_256.obj"
#define N 16*16*16

// Globals
float4 *p;
float3 *v, *f, *reactor,*r_GPU0, *r_GPU1;
float4 *p_GPU0, *p_GPU1;

__device__ float3 getMagForce(float4 p0, float3 v0, float3 dl_tail, float3 dl_head, float I){
    float3 dB, dl;
    dl.x = dl_head.x - dl_tail.x;
    dl.y = dl_head.y - dl_tail.y;
    dl.z = dl_head.z - dl_tail.z;

    float rx = p0.x - dl_tail.x;
    float ry = p0.y - dl_tail.y;
    float rz = p0.z - dl_tail.z;

    float r2 = rx*rx + ry*ry + rz*rz;
    float inv_r2 = 1 / r2;
    float inv_r = 1 / sqrtf(r2);
    float3 rhat = {rx * inv_r, ry * inv_r, rz * inv_r};

    float gamma = I;
    dB.x = gamma * (dl.y * rhat.z - dl.z * rhat.y) * inv_r2;
    dB.y = gamma * (dl.z * rhat.x - dl.x * rhat.z) * inv_r2;
    dB.z = gamma * (dl.x * rhat.y - dl.y * rhat.x) * inv_r2;

    return dB;
}

__global__ void getForcesMag(float4 *g_pos, float3 *vel, float3 *force, int offset, float3 *g_reactor){
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    if (id >= N) return; // Early exit if id is out of bounds

    float3 total_force = {0.0f, 0.0f, 0.0f};
    float3 B = {0.0f, 0.0f, 0.0f};
    float3 dB, dl_tail, dl_head;
    float4 posMe = g_pos[id + offset];
    float3 velMe = vel[id];
    
    __shared__ float3 shared_r[BLOCK];

    for (int k = 0; k < SHAPE_CT; k++) {
        if (threadIdx.x < SHAPE_SIZE) {  // Reduce shared memory usage
            shared_r[threadIdx.x] = g_reactor[threadIdx.x + blockDim.x * k];
        }
        __syncthreads();

        for (int j = 1; j <= SHAPE_SIZE; j++) {
            dl_tail = shared_r[(j - 1)];
            dl_head = shared_r[(j % SHAPE_SIZE)];
            dB = getMagForce(posMe, velMe, dl_tail, dl_head, 1.0);  // current[i] = 1

            B.x += dB.x;
            B.y += dB.y;
            B.z += dB.z;
        }
    }

    total_force.x = (velMe.y * B.z - velMe.z * B.y);
    total_force.y = (velMe.z * B.x - velMe.x * B.z);
    total_force.z = (velMe.x * B.y - velMe.y * B.x);

    force[id] = total_force;
}