#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NR_NEUTRONS 8
#define NR_ELECTRONS 8
#define NR_PROTONS 8

#define MASS_PROTON 1.007276
#define MASS_NEUTRON 1.008664
#define MASS_ELECTRON 5.485799e-4

#define BLOCK 256

#define XWindowSize 2500
#define YWindowSize 2500

#define DRAW 10
#define DAMP 1.0

#define DT 0.001
#define STOP_TIME 10.0

#define G 6.67408E-11
#define H 1.0

#define EYE 8.5
#define FAR 80.0

#define SHAPE_CT 24
#define SHAPE_SIZE 256
#define PATH "./objects/Tokamak_256.obj"
#define N 16*16*16

// Globals
float4 *p;
float3 *v, *f, *reactor,*r_GPU0, *r_GPU1;
float4 *p_GPU0, *p_GPU1;

__global__ void moveBodies(float4 *g_pos, float4 *d_pos, float3 *vel, float3 *force, int offset) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    if (id < N) {
        float inv_mass = 1.0f / d_pos[id].w;
        float damped_vel_x = vel[id].x * DAMP;
        float damped_vel_y = vel[id].y * DAMP;
        float damped_vel_z = vel[id].z * DAMP;

        // Update velocity using precomputed inverse mass and damped velocity
        vel[id].x = vel[id].x + ((force[id].x - damped_vel_x) * inv_mass) * DT;
        vel[id].y = vel[id].y + ((force[id].y - damped_vel_y) * inv_mass) * DT;
        vel[id].z = vel[id].z + ((force[id].z - damped_vel_z) * inv_mass) * DT;

        // Update positions
        d_pos[id].x += vel[id].x * DT;
        d_pos[id].y += vel[id].y * DT;
        d_pos[id].z += vel[id].z * DT;

        // Store updated positions in global memory
        g_pos[id + offset].x = d_pos[id].x;
        g_pos[id + offset].y = d_pos[id].y;
        g_pos[id + offset].z = d_pos[id].z;
    }
}