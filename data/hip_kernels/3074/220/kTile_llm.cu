#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate global thread ID
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total number of threads
    const int numThreads = blockDim.x * gridDim.x;
    const unsigned int numEls = tgtWidth * tgtHeight;

    // Loop through elements processed by this thread
    for (uint i = idx; i < numEls; i += numThreads) {
        // Determine position in target and corresponding source
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        
        // Copy element from source to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}