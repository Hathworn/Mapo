#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x;
    ii += (threadIdx.y + blockDim.y * blockIdx.y) * blockDim.x * gridDim.x;
    
    // Early exit for threads beyond elements count
    if (ii >= no_elements) return;
    
    // Use registers to avoid redundant calculations within loop
    const int base_idx = ii;
    const int sf_sq = scale_factor * scale_factor;

    for (int i = 0; i < sf_sq; i++){
        int off_x = i / scale_factor;
        int off_y = i % scale_factor;
        
        // Compute index in reduced grid
        int ipidx = translate_idx_inv(base_idx, d1, d2, d3, scale_factor, off_x, off_y);

        // Accumulate results
        gradInput_data[ii] += gradOutput_data[ipidx];
    }
}