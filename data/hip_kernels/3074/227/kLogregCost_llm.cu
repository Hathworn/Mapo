#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // If labelp is not maxp, set correctProbs[tx] to 0; otherwise, calculate correctProbs[tx].
        if (labelp != maxp) {
            correctProbs[tx] = 0.0f;
        } else {
            // Use float comparison and count the number of maximum probabilities.
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp);
            }
            // Safely handle numMax being zero, though it should be one.
            correctProbs[tx] = numMax > 0 ? 1.0f / float(numMax) : 0.0f; 
        }
    }
}