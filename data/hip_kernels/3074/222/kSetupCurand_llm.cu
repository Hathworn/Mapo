#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Use blockDim.x for dynamic thread handling
    const uint tidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize each state with a unique sequence number
    hiprand_init(seed, tidx, 0, &state[tidx]);
}