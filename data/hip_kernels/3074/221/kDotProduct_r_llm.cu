#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;
    if (eidx < numElements) {
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            sum += a[eidx] * b[eidx];
        }
        shmem[threadIdx.x] = sum;
    } else {
        shmem[threadIdx.x] = 0.0f;
    }
    __syncthreads();

    // Optimize reduction
    for (uint stride = DP_BLOCKSIZE / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write result to the global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}