#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel( const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    float tmp = 0.0f;
    
    // Unroll loop for efficiency
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Use a single warp for reduction if possible
    if (idx < 32) {
        for (int i = idx + 32; i < blockDim.x; i += 32) {
            reduction_buffer[idx] += reduction_buffer[i];
        }
        for (int offset = 16; offset > 0; offset /= 2) {
            reduction_buffer[idx] += __shfl_down_sync(0xffffffff, reduction_buffer[idx], offset);
        }
    }

    if (idx == 0) {
        reduction_buffer[0] = tmp = reduction_buffer[0];
    }
    __syncthreads();

    // Compute gradient
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}