#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use a single thread to divide the accuracy by N
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}