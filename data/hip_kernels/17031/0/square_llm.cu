#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(int *array, int n) {
    // Use combined identification for threads
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        array[tid] = array[tid] * array[tid];
    }
}