#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gMaxPoolingForward(float* out, int outRows, int outCols, float* in, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid >= outRows * outCols)
        return;

    int rowId = tid / outRows;
    int colId = tid % outRows;

    // Align base pointers for input and mask
    float* b = in + (rowId * inCols) + (colId * width);
    float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;

    // Adjust width for the last pooling window
    if(colId == outRows - 1) {
        width = lastWidth;
    }

    float currentMax = b[0] * localMask[0];
    
    #pragma unroll 4 // Suggest compiler to unroll the loop
    for(int i = 1; i < width; ++i) {
        float val = b[i] * localMask[i];
        if(val > currentMax) {
            currentMax = val;
        }
    }

    // Write the result to the output
    out[rowId * outCols + colId] = currentMax;
}