#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gScatterAdd(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure idx is within bounds of sparseSize
    if (idx < sparseSize) {
        int sparseIdx = sparseIndices[idx] + offset;
        
        // Check bounds for denseData updates
        if (sparseIdx >= 0 && sparseIdx < denseSize) {
            atomicAdd(&denseData[sparseIdx], sparseData[idx]); // Use atomicAdd for thread safety
        }
    }
}