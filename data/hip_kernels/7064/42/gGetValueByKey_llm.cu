#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gGetValueByKey(float* d_in, float* d_out, int* indeces, int n) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    // Access memory in a coalesced manner by assigning the index first
    if(tid < n) {
        int index = indeces[tid];
        
        // Prefetch input data to avoid redundant indexing
        float value = d_in[index];
        
        // Store pre-fetched data in the output
        d_out[tid] = value;
    }
}