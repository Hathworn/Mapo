#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gSetSparse(float* out, const size_t* indices, const float* values, int length) {
    // Calculate the unique global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Iterate over all elements this thread is responsible for
    for(int i = index; i < length; i += blockDim.x * gridDim.x) {
        // Perform the sparse setting
        out[indices[i]] = values[i];
    }
}