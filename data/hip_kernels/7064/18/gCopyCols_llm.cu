#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gCopyCols(float* out, const float* in, size_t rows, size_t colsIn, const size_t* sourceColIdx, size_t colsOut) {
    // Calculate the initial row and column index based on grid and block size
    int j = blockIdx.x + blockIdx.y * gridDim.x;
    int i = threadIdx.x;

    // Loop over rows in a coalesced manner
    while (j < rows) {
        const float* rowIn = in + j * colsIn;
        float* rowOut = out + j * colsOut;

        // Loop over columns using threads
        while (i < colsOut) {
            rowOut[i] = rowIn[sourceColIdx[i]];
            i += blockDim.x; // Move thread id to next stride
        }
        
        j += gridDim.x * gridDim.y; // Move block id to next stride
    }
}