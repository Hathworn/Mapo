#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gArgmax(float* out, const float* data, size_t rows, size_t cols) {
    // Calculate global thread index
    size_t row = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread operates within bounds
    if (row < rows) {
        size_t startInd = row * cols;
        float maxScore = -FLT_MAX;  // Use of FLT_MAX for better flexibility
        size_t maxInd = 0;

        // Iterate over columns to find max score and index
        for (size_t col = 0; col < cols; ++col) {
            size_t ind = startInd + col;
            float score = data[ind];
            if (score > maxScore) {
                maxScore = score;
                maxInd = col;
            }
        }
        out[row] = maxInd; // Store result in the output array
    }
}