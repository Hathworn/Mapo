#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableLogit(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

// Kernel function optimized to use efficient grid-stride loop
__global__ void gHighwayBackward(float* out1, float* out2, float* outt, const float* in1, const float* in2, const float* t, const float* adj, size_t length) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = gridDim.x * blockDim.x;
    for(size_t i = index; i < length; i += stride) {
        float sigma = stableLogit(t[i]);
        out1[i] = sigma * adj[i];  // Compute output 1
        out2[i] = (1.f - sigma) * adj[i];  // Compute output 2
        outt[i] = sigma * (1.f - sigma) * (in1[i] - in2[i]) * adj[i];  // Compute output t
    }
}