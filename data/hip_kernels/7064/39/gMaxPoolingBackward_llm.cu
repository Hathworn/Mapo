#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gMaxPoolingBackward(float* adj, int adjRows, int adjCols, float* in, float* adjIn, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid >= adjRows * adjCols)
        return;

    int rowId = tid / adjRows;
    int colId = tid % adjRows;

    // Use shared memory to load input data for current rowId
    extern __shared__ float sharedIn[];
    float* b = in + (rowId * inCols) + (colId * width);
    for(int i = 0; i < width; i++) {
        sharedIn[i] = b[i];
    }
    __syncthreads();

    if(colId == adjRows - 1) {
        width = lastWidth;
    }

    float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;
    size_t currentMaxIdx = 0;
    // Optimize to reduce thread divergence in loop
    float maxVal = sharedIn[0] * localMask[0];
    for(int i = 1; i < width; ++i) {
        float value = sharedIn[i] * localMask[i];
        if(value > maxVal) {
            maxVal = value;
            currentMaxIdx = i;
        }
    }

    // Use atomic operation to prevent race condition
    atomicAdd(&adjIn[(rowId * inCols) + (colId * width) + currentMaxIdx], adj[rowId + (colId * adjCols)]);
}