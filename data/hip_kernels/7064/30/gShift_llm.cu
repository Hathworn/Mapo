#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gShift(float* out, const float* in, int length, int offset) {
    // Calculate the global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Loop over elements with a stride of total number of threads
    for(int idx = index; idx < length; idx += blockDim.x * gridDim.x) {
        // Check boundary conditions
        if(idx - offset < 0 || idx - offset >= length)
            out[idx] = 0;
        else
            out[idx] = in[idx - offset];
    }
}