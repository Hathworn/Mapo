#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gCopyRows(float* out, const float* in, size_t cols, const size_t* sourceRowIdx, size_t rows) {
    // Calculate row index handled by the current block
    int j = blockIdx.x + blockIdx.y * gridDim.x;

    // Calculate column index handled by the current thread within the block
    int i = threadIdx.x + blockDim.x * threadIdx.y;

    if (j < rows) {
        size_t dstId = j;
        size_t srcId = sourceRowIdx[j];

        float* rowOut = out + dstId * cols;
        const float* rowIn = in + srcId * cols;

        if (i < cols) {
            rowOut[i] = rowIn[i];
        }
    }
}
```
