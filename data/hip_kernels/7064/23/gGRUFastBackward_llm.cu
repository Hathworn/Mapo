#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableLogit(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gGRUFastBackward(float* outState, float* outXW, float* outSU, float* outB,
                                 const float* state, const float* xW, const float* sU,
                                 const float* b, const float* mask, const float* adj,
                                 size_t rows, size_t cols, bool final) {
    int j = blockIdx.x;
    if(j < rows) {
        float m = !mask || mask[j];

        float* rowOutState = outState + j * cols;
        float* rowOutXW = outXW + j * cols * 3;
        float* rowOutSU = outSU + j * cols * 3;

        const float* rowState = state + j * cols;
        const float* rowXW = xW + j * cols * 3;
        const float* rowSU = sU + j * cols * 3;
        const float* rowAdj = adj + j * cols;

        int i = threadIdx.x;
        if(i < cols) {
            int k = i + cols;
            int l = i + 2 * cols;

            float r = stableLogit(rowXW[i] + rowSU[i] + b[i]);
            float z = stableLogit(rowXW[k] + rowSU[k] + b[k]);

            float h;
            if(final)
                h = tanhf(rowXW[l] + (rowSU[l] + b[l]) * r);
            else
                h = tanhf(rowXW[l] + rowSU[l] * r + b[l]);

            float adj = rowAdj[i];

            float t = (1 - z) * (1 - h * h);

            // df/ds
            if(outState)
                rowOutState[i] += (m * z - m + 1) * adj;

            // df/d(xW_r) ...
            float dfdxW_r = m * r * (1 - r) * t * adj;
            dfdxW_r *= final ? rowSU[l] + b[l] : rowSU[l];
            if(outXW)
                rowOutXW[i] += dfdxW_r;
            if(outSU)
                rowOutSU[i] += dfdxW_r;
            if(outB)
                atomicAdd(outB + i, dfdxW_r);

            // df/d(xW_z) ...
            float dfdxW_z = m * (1 - z) * z * (rowState[i] - h) * adj;
            if(outXW)
                rowOutXW[k] += dfdxW_z;
            if(outSU)
                rowOutSU[k] += dfdxW_z;
            if(outB)
                atomicAdd(outB + k, dfdxW_z);

            // df/d(xW_x) ...
            float dfdxW_x = m * t * adj;
            if(outXW)
                rowOutXW[l] += dfdxW_x;
            if(outSU)
                rowOutSU[l] += dfdxW_x * r;
            if(outB)
                atomicAdd(outB + l, final ? dfdxW_x * r : dfdxW_x);
        }
    }
}