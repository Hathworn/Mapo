#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gLogSoftmaxGrad(float* grad, const float* adj, const float* val, const int rows, const int cols) {
    int j = blockIdx.x + blockIdx.y * gridDim.x;
    if(j < rows) {
        extern __shared__ float _sharedMem[];
        float* _sum = _sharedMem;

        float* gradRow = grad + j * cols;
        const float* adjRow = adj + j * cols;
        const float* valRow = val + j * cols;

        // Initialize shared memory
        float threadSum = 0.0;
        for(int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
            threadSum += adjRow[tid];
        }
        _sum[threadIdx.x] = threadSum;

        // Reduce to get the sum across the block
        __syncthreads();
        for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
            if(threadIdx.x < offset) {
                _sum[threadIdx.x] += _sum[threadIdx.x + offset];
            }
            __syncthreads();
        }

        // Update gradients
        for(int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
            gradRow[tid] += adjRow[tid] - (expf(valRow[tid]) * _sum[0]);
        }
    }
}