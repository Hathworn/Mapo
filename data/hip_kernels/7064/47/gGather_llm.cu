#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gGather(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use early return pattern to improve warp efficiency
    if (idx >= sparseSize || sparseIndices[idx] < -offset || sparseIndices[idx] + offset >= denseSize) 
        return;

    sparseData[idx] = denseData[sparseIndices[idx] + offset];
}