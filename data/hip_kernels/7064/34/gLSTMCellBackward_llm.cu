#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableLogit(float x) {
    return x >= 0 ? 1.0 / (1.0 + expf(-x)) : expf(x) / (1.0 + expf(x));
}

__global__ void gLSTMCellBackward(float* outCell, float* outXW, float* outSU, float* outB, const float* cell, const float* xW, const float* sU, const float* b, const float* mask, const float* adj, size_t rows, size_t cols) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the index is within bounds
    if (j >= rows) return;

    float m = !mask || mask[j];

    float* rowOutCell = outCell + j * cols;
    float* rowOutXW = outXW + j * cols * 4;
    float* rowOutSU = outSU + j * cols * 4;

    const float* rowCell = cell + j * cols;
    const float* xWrow = xW + j * cols * 4;
    const float* sUrow = sU + j * cols * 4;
    
    const float* rowAdj = adj + j * cols;

    int i = threadIdx.x;

    if (i < cols) {
        float gf = stableLogit(xWrow[i] + sUrow[i] + b[i]);
        
        int k = i + cols;
        float gi = stableLogit(xWrow[k] + sUrow[k] + b[k]);
        
        int l = i + 2 * cols;
        float gc = tanhf(xWrow[l] + sUrow[l] + b[l]);
        
        float adj = rowAdj[i];
        
        // dc/dc_{t-1}
        if (outCell)
            rowOutCell[i] += (m * gf - m + 1) * adj;
        
        // dc/d(b_f)
        float dcdxf = m * rowCell[i] * gf * (1 - gf) * adj;
        if (outXW)
            rowOutXW[i] += dcdxf;
        if (outSU)
            rowOutSU[i] += dcdxf;
        if (outB)
            atomicAdd(outB + i, dcdxf);
        
        // dc/d(b_i)
        float dcdb_i = m * gc * gi * (1 - gi) * adj;
        if (outXW)
            rowOutXW[k] += dcdb_i;
        if (outSU)
            rowOutSU[k] += dcdb_i;
        if (outB)
            atomicAdd(outB + k, dcdb_i);
        
        // dc/d(b_c)
        float dcdxc = m * gi * (1 - gc * gc) * adj;
        if (outXW)
            rowOutXW[l] += dcdxc;
        if (outSU)
            rowOutSU[l] += dcdxc;
        if (outB)
            atomicAdd(outB + l, dcdxc);
    }
}