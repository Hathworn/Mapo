#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableLogit(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gHighwayForward(float* out, const float* in1, const float* in2, const float* t, size_t length) {
    // Calculate the global index for each thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Step through the array with stride equal to total threads
    for(int i = index; i < length; i += blockDim.x * gridDim.x) {
        float sigma = stableLogit(t[i]);
        out[i] = in1[i] * sigma + in2[i] * (1.f - sigma);
    }
}