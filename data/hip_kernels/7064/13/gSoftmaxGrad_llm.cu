#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gSoftmaxGrad(float* grad, const float* adj, const float* val, const int rows, const int cols) {
    for(int bid = 0; bid < rows; bid += gridDim.x) {
        int j = bid + blockIdx.x;
        if(j < rows) {
            extern __shared__ float _share[];
            float* _sum = _share;

            float* gradRow = grad + j * cols;
            const float* adjRow = adj + j * cols;
            const float* valRow = val + j * cols;
            _sum[threadIdx.x] = 0.0;

            // Optimize reduction using a single loop
            for(int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
                _sum[threadIdx.x] += valRow[tid] * adjRow[tid];
            }

            __syncthreads();
            
            // Reduce within shared memory
            for(int s = blockDim.x / 2; s > 0; s >>= 1) {
                if(threadIdx.x < s) {
                    _sum[threadIdx.x] += _sum[threadIdx.x + s];
                }
                __syncthreads();
            }

            // Update grad using optimized sum calculation
            for(int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
                float val = valRow[tid] * (adjRow[tid] - _sum[0]);
                if(val != 0.0) {
                    gradRow[tid] += val;
                }
            }
        }
    }
}