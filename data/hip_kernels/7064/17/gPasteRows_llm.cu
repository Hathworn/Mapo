#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gPasteRows(float* out, const float* in, size_t cols, const size_t* targetRowIdx, size_t rows) {
    // Calculate the global row index handled by this block
    int j = blockIdx.x + gridDim.x * blockIdx.y;
    if(j < rows) {
        size_t dstId = targetRowIdx[j];
        size_t srcId = j;

        float* rowOut = out + dstId * cols;
        const float* rowIn = in + srcId * cols;

        // Use a single loop to handle row copying with atomicAdd
        for(int i = threadIdx.x; i < cols; i += blockDim.x) {
            atomicAdd(rowOut + i, rowIn[i]);
        }
    }
}