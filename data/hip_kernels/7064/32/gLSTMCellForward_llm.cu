#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableLogit(float x) {
    if (x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gLSTMCellForward(float* out, const float* cell, const float* xW, const float* sU, const float* b, const float* mask, size_t rows, size_t cols) {
    // Calculate global and block indices
    int j = blockIdx.x + blockIdx.y * gridDim.x;
    int i = threadIdx.x;

    if (j < rows && i < cols) {
        float m = !mask || mask[j];

        // Calculate pointers to the current row
        float* rowOut = out + j * cols;
        const float* rowCell = cell + j * cols;
        const float* xWrow = xW + j * cols * 4;
        const float* sUrow = sU + j * cols * 4;

        // Calculating gate activations
        float gf = stableLogit(xWrow[i] + sUrow[i] + b[i]);
        float gi = stableLogit(xWrow[i + cols] + sUrow[i + cols] + b[i + cols]);
        float gc = tanhf(xWrow[i + 2 * cols] + sUrow[i + 2 * cols] + b[i + 2 * cols]);

        // Calculate output
        float cout = gf * rowCell[i] + gi * gc;
        rowOut[i] = m * cout + (1 - m) * rowCell[i];
    }
}