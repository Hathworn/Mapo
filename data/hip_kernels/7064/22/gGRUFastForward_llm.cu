#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline float stableLogit(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gGRUFastForward(float* out, const float* state, const float* xW, const float* sU, const float* b, const float* mask, size_t rows, size_t cols, bool final) {
    int j = blockIdx.x + blockIdx.y * gridDim.x; // Combine grid dimensions for more flexibility
    if(j < rows) {
        float m = !mask || mask[j];
        float* rowOut = out + j * cols;
        const float* rowState = state + j * cols;
        const float* xWrow = xW + j * cols * 3;
        const float* sUrow = sU + j * cols * 3;
        
        int stride = blockDim.x * blockDim.y; // Ensure wide coverage of threads
        for(int i = threadIdx.x + threadIdx.y * blockDim.x; i < cols; i += stride) {
            float r = stableLogit(xWrow[i] + sUrow[i] + b[i]);
            
            int k = i + cols;
            float z = stableLogit(xWrow[k] + sUrow[k] + b[k]);
            
            int l = i + 2 * cols;
            float h;
            if(final)
                h = tanhf(xWrow[l] + (sUrow[l] + b[l]) * r);
            else
                h = tanhf(xWrow[l] + sUrow[l] * r + b[l]);
            
            float outVal = (1.0f - z) * h + z * rowState[i];
            rowOut[i] = m * outVal + (1 - m) * rowState[i];
        }
    }
}