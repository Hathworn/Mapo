#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableLogit(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gLSTMOutputForward(float* out, const float* cell, const float* xW, const float* sU, const float* b, size_t rows, size_t cols) {
    int j = blockIdx.x + blockIdx.y * gridDim.x; // Unique ID for the row processed by each block
    if(j < rows) {
        float* rowOut = out + j * cols;
        const float* rowCell = cell + j * cols;
        const float* xWrow = xW + j * cols * 4;
        const float* sUrow = sU + j * cols * 4;

        for(int i = threadIdx.x; i < cols; i += blockDim.x) { // Loop over columns using threads
            int k = i + 3 * cols;
            float go = stableLogit(xWrow[k] + sUrow[k] + b[k]);
            rowOut[i] = go * tanhf(rowCell[i]);
        }
    }
}
```
