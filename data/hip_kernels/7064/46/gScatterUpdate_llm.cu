#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gScatterUpdate(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < sparseSize) { // Combine condition check into a single if statement
        int denseIdx = sparseIndices[idx] + offset; // Precompute dense index
        if(denseIdx >= 0 && denseIdx < denseSize) { // Simplified boundary check
            denseData[denseIdx] = sparseData[idx];
        }
    }
}