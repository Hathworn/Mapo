#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableLogit(float x) {
    if (x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gLSTMOutputBackward(float* outCell, float* outXW, float* outSU, float* outB, const float* cell, const float* xW, const float* sU, const float* b, const float* adj, size_t rows, size_t cols) {
    int j = blockIdx.x + blockIdx.y * gridDim.x;
    // Ensure grid size is sufficient to process all rows
    if (j < rows) {
        // Calculate global memory locations for this row
        float* rowOutCell = outCell + j * cols;
        float* rowOutXW = outXW + j * cols * 4;
        float* rowOutSU = outSU + j * cols * 4;

        const float* rowCell = cell + j * cols;
        const float* xWrow = xW + j * cols * 4;
        const float* sUrow = sU + j * cols * 4;

        const float* rowAdj = adj + j * cols;

        int i = threadIdx.x;
        // Optimize loop by relying on parallel threads
        while (i < cols) {
            int k = i + 3 * cols;
            float go = stableLogit(xWrow[k] + sUrow[k] + b[k]);

            float t = tanhf(rowCell[i]);
            float adjVal = rowAdj[i];

            // dc/dc_{t-1}
            if (outCell)
                rowOutCell[i] += go * (1 - t * t) * adjVal;

            // dc/d(b_o) = dc/d(xW_f) ...
            float dcdxo = t * go * (1 - go) * adjVal;
            if (outXW)
                rowOutXW[k] += dcdxo;
            if (outSU)
                rowOutSU[k] += dcdxo;
            if (outB)
                atomicAdd(outB + k, dcdxo);

            i += blockDim.x; // Move to the next element within the row
        }
    }
}