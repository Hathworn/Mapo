#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gPasteCols(float* out, const float* in, size_t rows, size_t colsOut, const size_t* targetColIdx, size_t colsIn) {
    int j = blockIdx.x + blockIdx.y * gridDim.x; // Calculate global row index
    if(j < rows) {
        const float* rowIn = in + j * colsIn;
        float* rowOut = out + j * colsOut;

        for(int tid = threadIdx.x; tid < colsIn; tid += blockDim.x) { // Optimize by eliminating extra outer loop
            if(tid < colsIn) {
                rowOut[targetColIdx[tid]] = rowIn[tid]; // Map input column to target output column
            }
        }
    }
}