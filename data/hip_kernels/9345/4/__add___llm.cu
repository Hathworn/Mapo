#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __add__(int *array, int *size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if idx is out of bounds
    if (idx >= *size) return;

    // Eliminate redundant operations
    int before = (idx + 1) % *size;
    int after = (idx == 0) ? (*size - 1) : (idx - 1);

    int temp = array[idx] + array[before] + array[after];
    
    __syncthreads(); // Synchronization barrier
    
    // Update array value
    array[idx] = temp;
}