#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelVacio(void) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int bDim = blockDim.x;

    // Reduced the condition to minimize thread divergence
    if (tid < 10) {
        printf("Data: %s Id Thread: %d Id block: %d Num threads block: %d\n", "helloWorld!", tid, bid, bDim);
    }
}