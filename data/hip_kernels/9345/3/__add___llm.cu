#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __add__(int *array, int *size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= *size) return; // Use >= for boundary check

    int temp = 0;
    int before = (idx + 1) % *size;
    int after = (idx - 1 + *size) % *size; // Adjusted to prevent negative index

    temp += array[idx];
    temp += array[before];
    temp += array[after];

    __syncthreads(); // Synchronize threads before writing
    array[idx] = temp;
}