#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ForwardLinear(float *A, float *W, float *b, int nRowsW, int nColsW, int nColsA, float *Z)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < nRowsW && col < nColsA)
    {
        float ZValue = 0.0f;

        // Utilize shared memory to improve memory access efficiency
        __shared__ float sharedA[NUM_THREADS_PER_BLOCK_X]; // Assuming sufficient shared memory

        for (int i = 0; i < nColsW; i += blockDim.x) 
        {
            if (i + threadIdx.x < nColsW) 
            {
                sharedA[threadIdx.x] = A[(i + threadIdx.x) * nColsA + col];
            }
            __syncthreads();

            for (int j = 0; j < blockDim.x && i + j < nColsW; ++j) 
            {
                ZValue += W[row * nColsW + i + j] * sharedA[j];
            }
            __syncthreads();
        }
        Z[row * nColsA + col] = ZValue + b[row];
    }
}