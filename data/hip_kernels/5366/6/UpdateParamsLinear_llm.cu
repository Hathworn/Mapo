#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UpdateParamsLinear(float *dZ, float *A, int nRowsdZ, int nColsdZ, int nRowsA, float lr, float *W, float *b)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < nRowsdZ && col < nRowsA)
    {
        float dWValue = 0;
        for (int i = 0; i < nColsdZ; i++)
        {
            dWValue += dZ[row * nColsdZ + i] * A[col * nColsdZ + i];
        }
        W[row * nRowsA + col] -= lr * dWValue / nColsdZ; // Simplified update

        if (col == 0) // Only execute for the first column
        {
            float dbValue = 0;
            for (int i = 0; i < nColsdZ; i++)
            {
                dbValue += dZ[row * nColsdZ + i];
            }
            b[row] -= lr * dbValue / nColsdZ; // Simplified update
        }
    }
}