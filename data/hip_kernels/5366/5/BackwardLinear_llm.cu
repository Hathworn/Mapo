#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BackwardLinear(float *dZ, float *W, int nColsW, int nRowsW, int nColsdZ, float *dA)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float dAValue = 0;

    // Check bounds before entering loop
    if (row < nColsW && col < nColsdZ)
    {
        // Use shared memory to optimize access to W array
        __shared__ float sharedW[blockDim.y][blockDim.x];
        
        for (int i = 0; i < nRowsW; i++)
        {
            sharedW[threadIdx.y][threadIdx.x] = W[i * nColsW + row];
            __syncthreads(); // Ensure all threads have loaded data
            
            dAValue += sharedW[threadIdx.y][threadIdx.x] * dZ[i * nColsdZ + col];
            __syncthreads(); // Ensure completion before next iteration
        }
        
        dA[row * nColsdZ + col] = dAValue;
    }
}