#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BackwardReLU(float* Z, float* dA, int nRowsdZ, int nColsdZ, float* dZ)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int totalElements = nRowsdZ * nColsdZ;

    // Ensure index is within bounds
    if (index < totalElements)
    {
        // Use ternary operator for concise conditional operation
        dZ[index] = (Z[index] >= 0) ? dA[index] : 0;
    }
}