#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BackwardCrossEntropy(float *output, float *labels, int nColsOutput, float *dOutput)
{
    // Use threadIdx.x for better parallelization and avoid using blockId.x in loop
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Add boundary check for col
    if (col < nColsOutput) {
        float label = labels[col];
        float out = output[col];

        // Simplify calculation and avoid redundant operations
        float temp = (label / out) - ((1.0f - label) / (1.0f - out));
        dOutput[col] = -temp;
    }
}