#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ForwardReLU(float* Z, int nRowsZ, int nColsZ, float* A)
{
    // Calculate the global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds and apply ReLU 
    if (index < nRowsZ * nColsZ)
    {
        // Use ternary operator for concise conditional assignment
        A[index] = Z[index] >= 0 ? Z[index] : 0;
    }
}