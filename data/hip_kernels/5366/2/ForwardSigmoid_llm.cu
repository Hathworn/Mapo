#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ForwardSigmoid(float* Z, int nRowsZ, int nColsZ, float* A)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Unroll the loop to improve performance
    for (int i = index; i < nRowsZ * nColsZ; i += stride)
    {
        A[i] = 1 / (1 + exp(-Z[i]));
    }
}