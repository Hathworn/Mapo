#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BackwardSigmoid(float* Z, float* dA, int nRowsdZ, int nColsdZ, float *dZ)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < nRowsdZ * nColsdZ)
    {
        // Precompute sigmoid to avoid redundant computation
        float sigmoid = 1 / (1 + expf(-Z[index]));
        dZ[index] = sigmoid * (1 - sigmoid) * dA[index];
    }
}