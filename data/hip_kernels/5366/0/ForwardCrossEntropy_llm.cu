#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ForwardCrossEntropy(float *output, float *labels, int nColsOutput, float *loss)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Use grid stride loop for robustness
    if (col < nColsOutput) { // Ensure within bounds
        float temp = -(labels[col] * logf(output[col]) + logf(1.0f - output[col]) * (1.0f - labels[col]));
        atomicAdd(loss, temp); // Accumulate loss safely
    }
}