#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ForwardSoftmax(float *Z, int nColsZ, float *sumExp, float *A)
{
    int row = threadIdx.x;
    int col = blockIdx.x;
    extern __shared__ float shared_sumExp[];

    // Initialize shared sumExp
    if (row == 0) shared_sumExp[col] = 0.0f;
    __syncthreads();

    // Reduce sum in shared memory
    atomicAdd(&shared_sumExp[col], exp(Z[row * nColsZ + col]));
    __syncthreads();

    // Write final result to global memory
    if (row == 0) atomicAdd(&sumExp[col], shared_sumExp[col]);
    __syncthreads();
    
    // Calculate softmax and store in A
    A[row * nColsZ + col] = exp(Z[row * nColsZ + col]) / shared_sumExp[col];
}