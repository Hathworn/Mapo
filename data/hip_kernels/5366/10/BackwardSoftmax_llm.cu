#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BackwardSoftmax(float *A, float *dA, int nColsdZ, float *dZ)
{
    // Use shared memory to reduce repeated global memory accesses
    __shared__ float sharedA[1024];
    __shared__ float shareddA[1024];

    int row = threadIdx.x;
    int col = blockIdx.x;

    // Load data into shared memory
    sharedA[row] = A[row * nColsdZ + col];
    shareddA[row] = dA[row * nColsdZ + col];

    __syncthreads();

    // Compute using shared memory
    dZ[row * nColsdZ + col] = shareddA[row] * sharedA[row] * (1 - sharedA[row]);
}