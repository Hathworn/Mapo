#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

__global__ void Vector_Addition(int *a, int *b, int *c, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread ID
    if (tid < n)                                      // Ensure within bounds of array
    {
        c[tid] = a[tid] + b[tid];                     // Perform vector addition
    }
}