#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA runtime

// Helper functions and utilities to work with CUDA

//Standard C library

#define subCOL 5248
#define COL 5248
#define ROW 358
#define WARPABLEROW 512
#define blocksize 256
#define subMatDim subCOL*WARPABLEROW
#define targetMatDim ROW * COL
__global__ void reduce5(int *g_idata, int *g_odata, int g_size)
{
    __shared__ int sdata[blocksize];

    // each thread loads one element from global to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    // Load elements into shared memory safely with checks
    if (i < g_size) {
        sdata[tid] = g_idata[i] + (i + blockDim.x < g_size ? g_idata[i + blockDim.x] : 0);
    } else {
        sdata[tid] = 0;
    }
    __syncthreads();

    // Perform reduction using shared memory
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Unrolled loop for last warp
    if (tid < 32) {
        volatile int *vsmem = sdata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}