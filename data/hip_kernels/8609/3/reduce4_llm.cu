#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define subCOL 5248
#define COL 5248
#define ROW 358
#define WARPABLEROW 512
#define blocksize 256
#define subMatDim subCOL*WARPABLEROW
#define targetMatDim ROW * COL

__global__ void reduce4(int *g_idata, int *g_odata, int g_size)
{
    __shared__ int sdata[blocksize]; // Shared memory for block-level reduction

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    // Load data from global memory with boundary check
    if (i < g_size)
    {
        sdata[tid] = g_idata[i] + ((i + blockDim.x < g_size) ? g_idata[i + blockDim.x] : 0);
    }
    else
    {
        sdata[tid] = 0;
    }
    __syncthreads();

    // Reduction within shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // Ensure all threads reach this point before proceeding
    }

    // Write the result of the reduction for this block to global memory
    if (tid == 0) 
    {
        g_odata[blockIdx.x] = sdata[0];
    }
}