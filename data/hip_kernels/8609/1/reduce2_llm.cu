#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA runtime

// Helper functions and utilities to work with CUDA

//Standard C library

#define subCOL 5248
#define COL 5248
#define ROW 358
#define WARPABLEROW 512
#define blocksize 256
#define subMatDim subCOL*WARPABLEROW
#define targetMatDim ROW * COL

__global__ void reduce2(int *g_idata, int *g_odata, int g_size)
{
    __shared__ int sdata[blocksize];

    // Each thread loads one element from global to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < g_size) {  // Check bounds
        sdata[tid] = g_idata[i];
    } else {
        sdata[tid] = 0;  // Avoid reading out of bounds
    }
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {  // Reduce using powers of two
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}