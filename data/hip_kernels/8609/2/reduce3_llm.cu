#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA runtime

// Helper functions and utilities to work with CUDA

//Standard C library

#define subCOL 5248
#define COL 5248
#define ROW 358
#define WARPABLEROW 512
#define blocksize 256
#define subMatDim subCOL*WARPABLEROW
#define targetMatDim ROW * COL

__global__ void reduce3(int *g_idata, int *g_odata, int g_size)
{
    __shared__ int sdata[blocksize];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + tid;
    
    // Load data from global to shared memory; check bounds
    sdata[tid] = (i < g_size) ? g_idata[i] : 0;
    __syncthreads();
    
    // Efficient reduction in shared memory using loop unrolling
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write block result to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}