#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define subCOL 5248
#define COL 5248
#define ROW 358
#define WARPABLEROW 512
#define blocksize 256
#define subMatDim subCOL*WARPABLEROW
#define targetMatDim ROW * COL

__global__ void reduce1(int *g_idata, int *g_odata, int g_size)
{
    __shared__ int sdata[blocksize];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data and check bounds
    int my_value = (i < g_size) ? g_idata[i] : 0;
    sdata[tid] = my_value;
    __syncthreads();

    // Unroll the loop by a factor of 2 for optimization
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}