#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Initialize histogram to 0
    for (int i = 0; i < 13; i++)
        hist[i * 32 + tx] = 0.0f;
    __syncthreads();

    // Precompute Gaussian weights
    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    if (tx < 15)
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));

    // Calculate pixel coordinates
    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;
    int px = xp & 15;
    int x = tx - px;

    // Load data into shared memory
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = xp + x;
        int yi = yp + y;
        xi = max(0, min(xi, w - 1));  // Clamp xi to valid range
        yi = max(0, min(yi, h - 1));  // Clamp yi to valid range
        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    // Gradient calculation and histogram binning
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = 16.0f * atan2f(dy, dx) / 3.1416f + 16.5f;
            bin = (bin == 32) ? 0 : bin;  // Wrap bin index
            float grad = sqrtf(dx * dx + dy * dy);
            hist[32 * (x - 1) + bin] += grad * gauss[x] * gauss[y];
        }
    }
    __syncthreads();

    // Reduce histogram
    if (tx < 32) {
        for (int offset = 4; offset > 0; offset /= 2) {
            for (int y = 0; y < offset; y++)
                hist[y * 32 + tx] += hist[(y + offset) * 32 + tx];
            __syncthreads();
        }
        hist[tx] += hist[32 + tx];
    }
    __syncthreads();

    // Smooth histogram
    if (tx < 32) {
        hist[32 + tx] = 6 * hist[tx] + 4 * (hist[(tx + 1) & 31] + hist[(tx + 31) & 31]) + (hist[(tx + 2) & 31] + hist[(tx + 30) & 31]);
    }
    __syncthreads();

    // Find peaks
    if (tx < 32) {
        float v = hist[32 + tx];
        hist[tx] = (v > hist[32 + ((tx + 1) & 31)] && v >= hist[32 + ((tx + 31) & 31)]) ? v : 0.0f;
    }
    __syncthreads();

    // Find dominant orientations
    if (tx == 0) {
        float maxval1 = 0.0f, maxval2 = 0.0f;
        int i1 = -1, i2 = -1;
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }

        // Compute peak location
        float peak1 = i1 + 0.5f * (hist[32 + ((i1 + 1) & 31)] - hist[32 + ((i1 + 31) & 31)]) / (2.0f * maxval1 - hist[32 + ((i1 + 1) & 31)] - hist[32 + ((i1 + 31) & 31)]);
        d_Orient[bx] = 11.25f * ((peak1 < 0.0f) ? peak1 + 32.0f : peak1);

        if (maxval2 < 0.8f * maxval1) {
            d_Orient[bx + maxPts] = i2;
        } else {
            float peak2 = i2 + 0.5f * (hist[32 + ((i2 + 1) & 31)] - hist[32 + ((i2 + 31) & 31)]) / (2.0f * maxval2 - hist[32 + ((i2 + 1) & 31)] - hist[32 + ((i2 + 31) & 31)]);
            d_Orient[bx + maxPts] = 11.25f * ((peak2 < 0.0f) ? peak2 + 32.0f : peak2);
        }
    }
}