#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;
    
    // Precompute some values outside the loops
    int h32 = h / 32;
    int warpWidth = 32;
    
    for (int y = 0; y < h32; y++) {
        for (int x = 0; x < w; x += warpWidth) {
            unsigned int val = minmax[y * w + x + tx];
            if (val) {
                int xOffset = (y * warpWidth) * w + x + tx;
                while (val && numPts < maxPts) { // Use while loop for efficiency
                    if (val & 0x1) {
                        ptrs[numPts * warpWidth + tx] = xOffset + (numPts % warpWidth) * w; // Optimize index computation
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}