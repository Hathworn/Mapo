#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Calculate global indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Compute size and position
    const int sz = height * pitch;
    const int p = threadIdx.z * sz + y * pitch + x;

    // Check bounds and perform computation
    if (x < width && y < height) {
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
    // Synchronization is not needed as threads within a block write to separate memory locations
}