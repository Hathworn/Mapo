#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Calculate global thread coordinates more efficiently
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x; // Optimize memory access calculation
    if (x < width && y < height)
    {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // Remove redundant __syncthreads(); no shared memory used
}