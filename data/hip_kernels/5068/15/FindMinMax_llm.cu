#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * blockDim.x + tx;
    const int y = blockIdx.y * 16;
    int p = y * pitch + x;

    // Initialize shared memory with boundary checks
    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        minvals[tx] = FLT_MAX;
        maxvals[tx] = -FLT_MAX;
    }

    // Process each row in the block
    for (int ty = 1; ty < 16; ty++) {
        p += pitch;
        if (x < width) {
            float val = d_Data[p];
            // Atomic min/max to reduce warp-level conflicts
            atomicMin(&minvals[tx], val);
            atomicMax(&maxvals[tx], val);
        }
    }
    __syncthreads();

    // Reduction to find the minimum and maximum in this block
    int stride = blockDim.x / 2;
    while (stride > 0) {
        if (tx < stride) {
            minvals[tx] = min(minvals[tx], minvals[tx + stride]);
            maxvals[tx] = max(maxvals[tx], maxvals[tx + stride]);
        }
        __syncthreads();
        stride /= 2;
    }

    // Write final result for this block to global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}