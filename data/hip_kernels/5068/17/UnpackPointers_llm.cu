#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;

    // Calculate the start and step for x to allow more threads to work on different x positions.
    for (int y = blockIdx.y; y < h/32; y += gridDim.y) { // Allow multiple blocks along y
        for (int x = threadIdx.x + blockIdx.x * blockDim.x; x < w; x += blockDim.x * gridDim.x) { // Spread x across threads
            unsigned int val = minmax[y * w + x];

            if (val) {
                for (int k = 0; k < 32; k++) {
                    if ((val & 0x1) && (numPts < maxPts)) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x;
                        numPts++;
                    }
                    // Move the bits of val to the right, to process the next point.
                    val >>= 1;
                }
            }
        }
    }
}