#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * blockDim.x + tx; // Use blockDim.x instead of 128
    const int y = blockIdx.y * blockDim.y; // Use blockDim.y instead of 16
    const int b = blockDim.x;
    int p = y * width + x;

    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        minvals[tx] = FLT_MAX;
        maxvals[tx] = -FLT_MAX;
    }

    for (int ty = 1; ty < blockDim.y; ty++) { // Replace the hardcoded 16 with blockDim.y for scalability
        p += width;
        if (x < width) {
            float val = d_Data[p];
            minvals[tx] = fminf(minvals[tx], val); // Use fminf for better performance
            maxvals[tx] = fmaxf(maxvals[tx], val); // Use fmaxf for better performance
        }
    }
    __syncthreads();

    for (int d = b / 2; d > 0; d >>= 1) {
        if (tx < d) {
            minvals[tx] = fminf(minvals[tx], minvals[tx + d]); // Use fminf for better performance
            maxvals[tx] = fmaxf(maxvals[tx], maxvals[tx + d]); // Use fmaxf for better performance
        }
        __syncthreads();
    }

    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr + 0] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
    __syncthreads();
}