#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int blockSize = 1024;

// Optimized global kernel function with a check for index bounds
__global__ void assign(uint32_t *x, uint32_t n) {
    // Calculate the global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use only the first thread to perform the assignment
    if (idx == 0) {
        x[0] = n;
    }
}