#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void divergence_test_ker()
{
    int idx = threadIdx.x;  // Cache thread index
    bool is_even = (idx % 2 == 0);  // Compute and store condition result

    // Minimize divergence by grouping similar branches
    if (is_even) {
        printf("threadIdx.x %d : This is an even thread.\n", idx);
    } else {
        printf("threadIdx.x %d : This is an odd thread.\n", idx);
    }
}