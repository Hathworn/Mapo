#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16  // Define tile size

__device__ float rowcol_dot(float *matrix_a, float *matrix_b, int row, int col, int N) {
    float val = 0;
    for (int k = 0; k < N; k++) {
        val += matrix_a[row * N + k] * matrix_b[col + k * N];
    }
    return val;
}

__global__ void matrix_mult_ker(float *matrix_a, float *matrix_b, float *output_matrix, int N) {
    __shared__ float tile_a[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tile_b[TILE_WIDTH][TILE_WIDTH];

    int tile_row = threadIdx.y;
    int tile_col = threadIdx.x;
    int row = blockIdx.y * blockDim.y + tile_row;
    int col = blockIdx.x * blockDim.x + tile_col;

    float value = 0.0f;

    for (int p = 0; p < N / TILE_WIDTH; p++) {
        // Load data into shared memory
        tile_a[tile_row][tile_col] = matrix_a[row * N + (p * TILE_WIDTH + tile_col)];
        tile_b[tile_row][tile_col] = matrix_b[(p * TILE_WIDTH + tile_row) * N + col];

        __syncthreads(); // Ensure all threads have written to shared memory

        // Perform block-based multiplication
        for (int k = 0; k < TILE_WIDTH; k++) {
            value += tile_a[tile_row][k] * tile_b[k][tile_col];
        }

        __syncthreads(); // Wait for all threads to complete the computation
    }

    if (row < N && col < N) {
        output_matrix[row * N + col] = value; // Write result to global memory
    }
}