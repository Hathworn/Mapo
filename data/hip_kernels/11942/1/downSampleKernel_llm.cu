#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void downSampleKernel(unsigned char * d_in, unsigned char * d_out, size_t skip) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x; // Global index calculation
    size_t offset = idx * skip * 3; // Compute offset once

    // Bounds check to prevent out-of-bounds access
    if (offset + 2 < skip * gridDim.x * blockDim.x * 3) {
        int px = d_in[offset] + d_in[offset + 1] + d_in[offset + 2]; // Use computed offset
        d_out[idx] = px / 3; // Store averaged value
    }
}