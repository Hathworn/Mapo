#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __INTELLISENSE__
void __syncthreads();
#endif

// image dimensions WIDTH & HEIGHT
#define WIDTH 256
#define HEIGHT 256

// Block width WIDTH & HEIGHT
#define BLOCK_W 16
#define BLOCK_H 16

// buffer to read image into
float image[HEIGHT][WIDTH];

// buffer for resulting image
float final[HEIGHT][WIDTH];

// prototype declarations
void load_image();
void call_kernel();
void save_image();

#define MAXLINE 128

float total, sobel;
hipEvent_t start_total, stop_total;
hipEvent_t start_sobel, stop_sobel;

__global__ void imageBlur(float *input, float *output, int width, int height) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (row < height && col < width && row > 0 && col > 0) {
        int numcols = WIDTH;

        // Use shared memory for block-wide operations
        __shared__ float sharedInput[BLOCK_H][BLOCK_W];

        int localRow = threadIdx.y;
        int localCol = threadIdx.x;

        // Load data into shared memory
        sharedInput[localRow][localCol] = input[row * numcols + col];
        __syncthreads();

        // Check boundaries for shared memory usage
        if (localRow > 0 && localRow < BLOCK_H - 1 && localCol > 0 && localCol < BLOCK_W - 1) {
            float blur = 0.2f * (
                sharedInput[localRow + 1][localCol] +     // up
                sharedInput[localRow][localCol - 1] +     // left
                sharedInput[localRow][localCol] +         // center
                sharedInput[localRow][localCol + 1] +     // right
                sharedInput[localRow - 1][localCol]       // down
            );

            output[row * numcols + col] = blur;
        }
    }
}