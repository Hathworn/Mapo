#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#define WIDTH 256
#define HEIGHT 256
#define BLOCK_W 16
#define BLOCK_H 16

float image[HEIGHT][WIDTH];
float final[HEIGHT][WIDTH];

void load_image();
void call_kernel();
void save_image();

#define MAXLINE 128

float total, sobel;
hipEvent_t start_total, stop_total;
hipEvent_t start_sobel, stop_sobel;

__global__ void sobelFilter(float *input, float *output, int width, int height) {

// Optimize index calculation
int col = threadIdx.x + blockIdx.x * blockDim.x;
int row = threadIdx.y + blockIdx.y * blockDim.y;
int numcols = WIDTH;

// Initialize gradients and threshold
float gradient_h = 0.0f;
float gradient_v = 0.0f;
const float thresh = 30.0f;

// Avoid out-of-bound errors
if (row < height && col < width && row > 0 && col > 0)
{
int x0 = input[(row - 1) * numcols + (col - 1)]; // leftup
int x1 = input[(row - 1) * numcols + col];       // up
int x2 = input[(row - 1) * numcols + (col + 1)]; // rightup
int x3 = input[row * numcols + (col - 1)];       // left
int x5 = input[row * numcols + (col + 1)];       // right
int x6 = input[(row + 1) * numcols + (col - 1)]; // leftdown
int x7 = input[(row + 1) * numcols + col];       // down
int x8 = input[(row + 1) * numcols + (col + 1)]; // rightdown

// Calculate gradients
gradient_h = (x0 * -1.0f) + (x2 * 1.0f) + (x3 * -2.0f) + (x5 * 2.0f) + (x6 * -1.0f) + (x8 * 1.0f);
gradient_v = (x0 * -1.0f) + (x1 * -2.0f) + (x3 * -1.0f) + (x6 * 1.0f) + (x7 * 2.0f) + (x8 * 1.0f);

// Use hypotf for better performance
float gradient = hypotf(gradient_h, gradient_v);

// Binarize the gradient
output[row * numcols + col] = (gradient >= thresh) ? 255.0f : 0.0f;
}
}