#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void resampleFeaturesKernel(double* u, double* v, double* d, double* vu, double* vv, double* vd, double* weights, double* randvals, int n_features, double* u_sampled, double* v_sampled, double* d_sampled, double* vu_sampled, double* vv_sampled, double* vd_sampled)
{
    int n_particles = blockDim.x;
    double interval = 1.0 / n_particles; // Precompute interval for efficiency

    for (int n = blockIdx.x; n < n_features; n += gridDim.x) {
        double r = randvals[n] + threadIdx.x * interval;
        int offset = n * n_particles;
        int idx = offset;

        // Use shared memory to cache weights for faster access (only if fits into shared memory)
        __shared__ double sharedWeights[/* specify size based on blockDim.x */];
        if (threadIdx.x == 0) {
            for (int i = 0; i < n_particles; ++i) {
                sharedWeights[i] = weights[offset + i];
            }
        }
        __syncthreads();

        double c = sharedWeights[0];
        while (r > c && idx < offset + n_particles) {
            c += sharedWeights[++idx - offset];
        }

        int idx_new = offset + threadIdx.x;
        u_sampled[idx_new] = u[idx];
        v_sampled[idx_new] = v[idx];
        d_sampled[idx_new] = d[idx];
        vu_sampled[idx_new] = vu[idx];
        vv_sampled[idx_new] = vv[idx];
        vd_sampled[idx_new] = vd[idx];
    }
}