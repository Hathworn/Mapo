#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void sumByReduction(volatile double* sdata, double mySum, const unsigned int tid)
{
    sdata[tid] = mySum;
    __syncthreads();

    // Optimize reduction with less synchronization
    if (tid < 128) sdata[tid] += sdata[tid + 128];
    __syncthreads();
    if (tid < 64) sdata[tid] += sdata[tid + 64];
    __syncthreads();

    if (tid < 32)
    {
        // Loop unrolling for last warp
        if (tid < 32) sdata[tid] += sdata[tid + 32];
        if (tid < 16) sdata[tid] += sdata[tid + 16];
        if (tid < 8) sdata[tid] += sdata[tid + 8];
        if (tid < 4) sdata[tid] += sdata[tid + 4];
        if (tid < 2) sdata[tid] += sdata[tid + 2];
        if (tid < 1) sdata[tid] += sdata[tid + 1];
    }
}

__global__ void computePdKernel(double* particle_pd, int particles_per_feature, int n_features, double* feature_pd)
{
    __shared__ double shmem[256];
    for (int n = blockIdx.x; n < n_features; n += gridDim.x)
    {
        int offset = n * particles_per_feature;
        double val = 0;
        // Coalesced access by using a stride
        for (int i = offset + threadIdx.x; i < offset + particles_per_feature; i += blockDim.x)
        {
            val += particle_pd[i];
        }
        
        sumByReduction(shmem, val, threadIdx.x);

        // Write the result from thread 0
        if (threadIdx.x == 0)
            feature_pd[n] = shmem[0] / particles_per_feature;
            
        __syncthreads();
    }
}