#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expandKernel(double* values, int n_original, int factor, double* expanded){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    // Precompute division as integer for efficiency
    for (; tid < n_original * factor; tid += stride){
        int idx = tid / factor;  // Use integer division
        expanded[tid] = values[idx];  
    }
}