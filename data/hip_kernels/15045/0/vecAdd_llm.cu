#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/**********************************************************
* @author  Pulkit Verma
* @email   technopreneur[dot]pulkit[at]gmail[dot]com
**********************************************************/

// The program takes two equal size vectors as input and outputs their vector sum

// Optimized vecAdd kernel function
__global__ void vecAdd(float *in1, float *in2, float *out, int len)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into registers to reduce memory access and ensure warp efficiency
    float val1 = (i < len) ? in1[i] : 0.0f;
    float val2 = (i < len) ? in2[i] : 0.0f;

    // Compute the output if within bounds
    if (i < len) {
        out[i] = val1 + val2;
    }
}