#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inverse_transform(float *in, float *out, int height, int width) {
    // Calculate global index and ensure it is within bounds
    int my_x = blockIdx.x * blockDim.x + threadIdx.x;
    if (my_x >= width) return;

    // Iterate through each element, transforming from frequency to time domain
    for (int k = 0; k < height; k++) {
        float realSum = 0.0f;

        #pragma unroll // Unroll the loop for better performance
        for (int t = 0; t < width; t++) {
            float angle = 2 * M_PI * (my_x * height + t) * (my_x * height + k) / height;
            realSum += in[my_x * height + t] * cos(angle);
        }
        out[my_x * height + k] = realSum / height; // Normalize by height
    }
}