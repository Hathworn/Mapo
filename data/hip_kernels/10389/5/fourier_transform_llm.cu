#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fourier_transform(float *in, float *out, int height, int width, int blockConfig) {
    // Use shared memory to reduce global memory accesses
    extern __shared__ float shared_in[];

    int my_x = blockIdx.x * blockDim.x + threadIdx.x;
    int threadIndex = threadIdx.x;
    float realSum = 0.0;

    // Load elements into shared memory
    for (int t = threadIndex; t < width * height; t += blockDim.x) {
        shared_in[t] = in[t];
    }
    __syncthreads();

    // Calculate Fourier transform using shared memory
    if (my_x < height * width) {
        for (int k = 0; k < height; k++) {
            realSum = 0.0;
            for (int t = 0; t < width; t++) {
                float angle = 2 * M_PI * (my_x * height + t) * (my_x * width + k) / height;
                realSum += shared_in[my_x * height + t] * cos(angle);
            }
            out[my_x * height + k] = realSum;
        }
    }
}