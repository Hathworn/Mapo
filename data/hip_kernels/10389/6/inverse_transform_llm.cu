#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inverse_transform(float *in, float *out, int height, int width) {
    // Compute global thread index
    int my_x = blockIdx.x * blockDim.x + threadIdx.x;

    // Check to ensure thread is within bounds
    if (my_x < width) {
        // Iterate through each element, going from frequency to time domain
        for (int k = 0; k < height; k++) {
            float realSum = 0.0f;
            // Iterate through the input element
            for (int t = 0; t < width; t++) {
                float angle = 2.0f * M_PI * (my_x * height + t) * (my_x * height + k) / height;
                realSum += in[my_x * height + t] * cosf(angle);
            }
            // Store the result
            out[my_x * height + k] = realSum / height;
        }
    }
}