#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fourier_transform(float *in, float *out, int height, int width, int blockConfig) {
    // Calculate the global index
    int my_x = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop through each element from frequency to time domain
    for (int k = 0; k < height; k++) {
        // Initialize real part sum
        float realSum = 0.0;

        // Iterate through the input elements
        for (int t = 0; t < width; t++) {
            // Calculate the angle and update the real sum
            float angle = 2 * M_PI * (my_x * height + t) * (my_x * width + k) / height;
            realSum += in[my_x * height + t] * cosf(angle);  // Use cosf for single precision
        }

        // Store the result in the output array
        out[my_x * height + k] = realSum;
    }
}