#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yuv2rgb_kernel(int img_size, unsigned char *img_r, unsigned char *img_g, unsigned char *img_b, unsigned char *img_y, unsigned char *img_u, unsigned char *img_v) {

    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < img_size) {
        // Load YUV values from global memory and convert U,V to centered format
        unsigned char y = img_y[i];
        double cb = static_cast<double>(img_u[i]) - 128;
        double cr = static_cast<double>(img_v[i]) - 128;

        // Compute RGB values using modified YUV formula
        img_r[i] = min(max(y + 1.402 * cr, 0.0), 255.0);
        img_g[i] = min(max(y - 0.344 * cb - 0.714 * cr, 0.0), 255.0);
        img_b[i] = min(max(y + 1.772 * cb, 0.0), 255.0);
    }
}