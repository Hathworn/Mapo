#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rgb2hsl_kernel(int img_size, unsigned char *img_r, unsigned char *img_g, unsigned char *img_b, float *img_h, float *img_s, unsigned char *img_l)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    
    if (i >= img_size) return; // Check bounds to prevent out-of-bounds access

    float H, S, L;
    float var_r = ( (float)img_r[i] / 255.0f ); // Convert RGB to [0,1]
    float var_g = ( (float)img_g[i] / 255.0f );
    float var_b = ( (float)img_b[i] / 255.0f );

    // Compute min/max using fminf/fmaxf for better performance
    float var_min = fminf(var_r, fminf(var_g, var_b));
    float var_max = fmaxf(var_r, fmaxf(var_g, var_b));
    float del_max = var_max - var_min;

    L = (var_max + var_min) * 0.5f;
    if (del_max == 0.0f) // This is a gray, no chroma
    {
        H = 0.0f;
        S = 0.0f;
    }
    else // Chromatic data
    {
        S = (L < 0.5f) ? (del_max / (var_max + var_min)) : (del_max / (2.0f - var_max - var_min));

        float del_r = (((var_max - var_r) / 6.0f) + (del_max / 2.0f)) / del_max;
        float del_g = (((var_max - var_g) / 6.0f) + (del_max / 2.0f)) / del_max;
        float del_b = (((var_max - var_b) / 6.0f) + (del_max / 2.0f)) / del_max;

        if (var_r == var_max)
            H = del_b - del_g;
        else if (var_g == var_max)
            H = (1.0f / 3.0f) + del_r - del_b;
        else // var_b == var_max
            H = (2.0f / 3.0f) + del_g - del_r;
    }

    if (H < 0.0f) H += 1.0f;
    if (H > 1.0f) H -= 1.0f;

    img_h[i] = H;
    img_s[i] = S;
    img_l[i] = (unsigned char)(L * 255.0f);
}