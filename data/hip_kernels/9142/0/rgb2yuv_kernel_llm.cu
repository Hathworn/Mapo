#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rgb2yuv_kernel(int img_size, unsigned char *img_r, unsigned char *img_g, unsigned char *img_b, unsigned char *img_y, unsigned char *img_u, unsigned char *img_v) {
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if within bounds
    if (i < img_size) {
        // Read RGB values
        int r = img_r[i];
        int g = img_g[i];
        int b = img_b[i];

        // Compute YUV with precomputed constants and store
        img_y[i] = static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);
        img_u[i] = static_cast<unsigned char>(-0.169f * r - 0.331f * g + 0.499f * b + 128.0f);
        img_v[i] = static_cast<unsigned char>(0.499f * r - 0.418f * g - 0.0813f * b + 128.0f);
    }
}