#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Early exit for out of bounds threads
    if (i >= h || j >= w) return;

    const int pos = i * w + j;
    
    // Avoid unnecessary memory fetches
    image[pos] = value;
}