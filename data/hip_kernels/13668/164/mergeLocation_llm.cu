#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate global thread index and check boundary in a single step.
    int ptidx;
    if ((ptidx = blockIdx.x * blockDim.x + threadIdx.x) < npoints) 
    {
        // Use registers for temporary values to optimize memory access.
        short2 loc = loc_[ptidx];
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}