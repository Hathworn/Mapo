#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Calculate unique global index for each thread
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Return early if indices are out of bounds
    if (i >= h || j >= w) return;

    // Precompute row offsets
    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Retrieve flow vector components
    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Compute target pixel coordinates (bottom-left corner)
    float cx = u_ * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v_ * time_scale + static_cast<float>(i) + 1.0f;

    // Convert float coordinates to integer indices
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Fetch source pixel value
    float value = src[image_row_offset + j];

    // Use a boundary check prior to atomic addition
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        // Perform atomic add to ensure thread safety
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}