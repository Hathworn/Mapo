#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Calculate the thread position
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a loop to handle multiple data points for better efficiency
    for (int i = pos; i < len; i += blockDim.x * gridDim.x)
    {
        d_res[i] = d_src[i] * scale;
    }
}