#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global thread indices
    int j = blockIdx.x * blockDim.x + threadIdx.x; // x-dimension (width)
    int i = blockIdx.y * blockDim.y + threadIdx.y; // y-dimension (height)

    // Check if within image bounds
    if (i < h && j < w) {
        const int pos = i * s + j;

        float scale = normalization_factor[pos];
        float invScale = (scale != 0.0f) ? (1.0f / scale) : 1.0f;

        // Apply normalization
        image[pos] *= invScale;
    }
}