#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];
        float scaledX = loc.x * scale; // Precalculate scaled x
        float scaledY = loc.y * scale; // Precalculate scaled y

        x[ptidx] = scaledX; // Assign scaled values
        y[ptidx] = scaledY;
    }
}