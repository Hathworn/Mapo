#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Precompute values to reduce recalculation
    float j_f = static_cast<float>(j);
    float i_f = static_cast<float>(i);

    // bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + j_f + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + i_f + 1.0f;

    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);
    float value = src[image_row_offset + j];

    float weight;

    // Combined boundary condition check to reduce redundant checks
    bool in_bounds_bl = (tx >= 0) && (tx < w) && (ty >= 0) && (ty < h);
    bool in_bounds_br = (tx + 1 >= 0) && (tx + 1 < w) && (ty >= 0) && (ty < h);
    bool in_bounds_ul = (tx >= 0) && (tx < w) && (ty - 1 >= 0) && (ty - 1 < h);
    bool in_bounds_ur = (tx + 1 >= 0) && (tx + 1 < w) && (ty - 1 >= 0) && (ty - 1 < h);

    // fill pixel containing bottom left corner
    if (in_bounds_bl)
    {
        weight = (1.0f - dx) * dy;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // fill pixel containing bottom right corner
    if (in_bounds_br)
    {
        weight = dx * dy;
        _atomicAdd(dst + ty * image_stride + (tx + 1), value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + (tx + 1), weight);
    }

    // fill pixel containing upper left corner
    if (in_bounds_ul)
    {
        weight = (1.0f - dx) * (1.0f - dy);
        _atomicAdd(dst + (ty - 1) * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + (ty - 1) * image_stride + tx, weight);
    }

    // fill pixel containing upper right corner
    if (in_bounds_ur)
    {
        weight = dx * (1.0f - dy);
        _atomicAdd(dst + (ty - 1) * image_stride + (tx + 1), value * weight);
        _atomicAdd(normalization_factor + (ty - 1) * image_stride + (tx + 1), weight);
    }
}