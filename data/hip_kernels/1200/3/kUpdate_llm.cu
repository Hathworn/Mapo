#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kUpdate(int nbSpx, float* clusters, float* accAtt_g)
{
    int cluster_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (cluster_idx < nbSpx) {
        int nbSpx2 = nbSpx * 2;
        int nbSpx3 = nbSpx * 3;
        int nbSpx4 = nbSpx * 4;
        int nbSpx5 = nbSpx * 5;
        int counter = __float2int_rd(accAtt_g[cluster_idx + nbSpx5]); // Use fast integer conversion

        if (counter != 0) {
            float invCounter = 1.0f / counter; // Calculate reciprocal once
            clusters[cluster_idx] = accAtt_g[cluster_idx] * invCounter;
            clusters[cluster_idx + nbSpx] = accAtt_g[cluster_idx + nbSpx] * invCounter;
            clusters[cluster_idx + nbSpx2] = accAtt_g[cluster_idx + nbSpx2] * invCounter;
            clusters[cluster_idx + nbSpx3] = accAtt_g[cluster_idx + nbSpx3] * invCounter;
            clusters[cluster_idx + nbSpx4] = accAtt_g[cluster_idx + nbSpx4] * invCounter;

            // Use a loop to reset accumulator for better scalability
            for (int i = 0; i <= 5; ++i) {
                accAtt_g[cluster_idx + i * nbSpx] = 0;
            }
        }
    }
}