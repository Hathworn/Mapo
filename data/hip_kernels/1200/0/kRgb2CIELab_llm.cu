#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kRgb2CIELab(const hipTextureObject_t texFrameBGRA, hipSurfaceObject_t surfFrameLab, int width, int height) {

    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;

    if (px < width && py < height) {
        uchar4 nPixel = tex2D<uchar4>(texFrameBGRA, px, py); // Fetch pixel

        // Normalize RGB values
        float _r = nPixel.z / 255.0f;
        float _g = nPixel.y / 255.0f;
        float _b = nPixel.x / 255.0f;

        // Convert RGB to XYZ
        float x = _r * 0.412453f + _g * 0.357580f + _b * 0.180423f;
        float y = _r * 0.212671f + _g * 0.715160f + _b * 0.072169f;
        float z = _r * 0.019334f + _g * 0.119193f + _b * 0.950227f;

        // Normalize for D65 illumination
        x /= 0.950456f;
        z /= 1.088754f;

        // Convert XYZ to CIELab
        float y3 = cbrtf(y);
        x = (x > 0.008856f) ? cbrtf(x) : (7.787f * x + 0.137931034f);
        y = (y > 0.008856f) ? y3 : (7.787f * y + 0.137931034f);
        z = (z > 0.008856f) ? cbrtf(z) : (7.787f * z + 0.137931034f);

        float l = (y > 0.008856f) ? (116.0f * y3 - 16.0f) : (903.3f * y);
        float a = (x - y) * 500.0f;
        float b = (y - z) * 200.0f;

        // Write back to surface object
        float4 fPixel;
        fPixel.x = l;
        fPixel.y = a;
        fPixel.z = b;
        fPixel.w = 0.0f;

        surf2Dwrite(fPixel, surfFrameLab, px * sizeof(float4), py); // Correct stride based on float4 size
    }
}