#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kInitClusters(const hipSurfaceObject_t surfFrameLab, float* clusters, int width, int height, int nSpxPerRow, int nSpxPerCol) {
    int centroidIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int nSpx = nSpxPerCol * nSpxPerRow;

    if (centroidIdx < nSpx) {
        // Precompute values outside of the loop for better performance
        int wSpx = width / nSpxPerRow;
        int hSpx = height / nSpxPerCol;
        
        int i = centroidIdx / nSpxPerRow;
        int j = centroidIdx % nSpxPerRow;
        
        // Use precalculated wSpx and hSpx values
        int x = j * wSpx + wSpx / 2;
        int y = i * hSpx + hSpx / 2;

        float4 color;
        // Use constant stride value for surface reads
        surf2Dread(&color, surfFrameLab, x * sizeof(float4), y);
        
        int baseIdx = centroidIdx; 
        // Store cluster values using a calculated base index
        clusters[baseIdx] = color.x;
        clusters[baseIdx + nSpx] = color.y;
        clusters[baseIdx + 2 * nSpx] = color.z;
        clusters[baseIdx + 3 * nSpx] = static_cast<float>(x);
        clusters[baseIdx + 4 * nSpx] = static_cast<float>(y);
    }
}