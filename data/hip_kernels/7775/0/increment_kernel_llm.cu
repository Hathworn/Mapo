#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void increment_kernel(int *g_data, int inc_value)
{
    // Calculate index within grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if idx is within bounds to avoid accessing out of range
    if (idx < gridDim.x * blockDim.x)
    {
        g_data[idx] += inc_value; // Perform increment
    }
}