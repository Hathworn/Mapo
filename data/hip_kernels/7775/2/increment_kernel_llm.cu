#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void increment_kernel(int *g_data, int inc_value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < gridDim.x * blockDim.x) // Ensure idx is within valid range
    {
        g_data[idx] += inc_value; // Simplified increment operation
    }
}