#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernal(char *buffer, long size, int *histo)
{
    __shared__ int temp[256];
    // Initialize shared memory with 0
    if (threadIdx.x < 256) {
        temp[threadIdx.x] = 0;
    }
    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;
    while (i < size)
    {
        atomicAdd(&temp[(int)buffer[i]], 1); // Cast buffer element to int
        i += offset;
    }
    __syncthreads();

    // Accumulate results from shared memory to global memory
    if (threadIdx.x < 256) {
        atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
    }
}