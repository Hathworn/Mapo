#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calibrate_fix2float(float *dst, const float* sA, const float* sB, float alpha, float beta, int height, int width, int threads) {
    int ri = blockIdx.x;
    int tid = threadIdx.x;
    int idx = tid + blockIdx.x * blockDim.x; // Calculate a unique index per thread

    // Calculate loop based on total possible threads per block
    if (idx < width) {
        float rscale = (sA[ri] == 0.0f) ? 1.0f : sA[ri];
        float *data = dst + width * ri;

        float temp = data[idx];
        float cscale = (sB[idx] == 0.0f) ? 255.0f : sB[idx];
        data[idx] = beta * temp + alpha * temp * rscale * cscale;
    }
}