#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void index_init(int* out_data, int h, int w) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride to use in loop for coalesced memory access
    for (int i = idx; i < h * w; i += stride) { // Use calculated stride in loop
        out_data[i] = i % w; // Directly calculate and assign w_id
    }
}