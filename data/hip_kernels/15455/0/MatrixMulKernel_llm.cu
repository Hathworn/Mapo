#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16

__global__ void MatrixMulKernel(float *M, float *N, float *P, int Width)
{
    // Use shared memory for tiled matrix multiplication
    __shared__ float Ms[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Ns[BLOCK_SIZE][BLOCK_SIZE];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    
    int Row = by * blockDim.y + ty;
    int Col = bx * blockDim.x + tx;

    float Pvalue = 0;

    for (int ph = 0; ph < Width / BLOCK_SIZE; ++ph)
    {
        // Collaborative loading of tiles
        if (Row < Width && (ph * BLOCK_SIZE + tx) < Width)
            Ms[ty][tx] = M[Row * Width + ph * BLOCK_SIZE + tx];
        else
            Ms[ty][tx] = 0;
        
        if (Col < Width && (ph * BLOCK_SIZE + ty) < Width)
            Ns[ty][tx] = N[(ph * BLOCK_SIZE + ty) * Width + Col];
        else
            Ns[ty][tx] = 0;
        
        __syncthreads();

        // Accumulate results
        for (int k = 0; k < BLOCK_SIZE; ++k)
            Pvalue += Ms[ty][k] * Ns[k][tx];
        
        __syncthreads();
    }

    // Write results
    if (Row < Width && Col < Width)
        P[Row * Width + Col] = Pvalue;
}