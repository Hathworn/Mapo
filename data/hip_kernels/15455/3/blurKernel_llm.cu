#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blurKernel(uchar3 *in, uchar3 *out, int w, int h)
{
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;

    if (Col < w && Row < h)
    {
        int pixValR = 0, pixValG = 0, pixValB = 0; // Separate channels
        int pixels = 0;

        for (int blurRow = -BLUR_SIZE; blurRow <= BLUR_SIZE; ++blurRow)
        {
            for (int blurCol = -BLUR_SIZE; blurCol <= BLUR_SIZE; ++blurCol)
            {
                int curRow = Row + blurRow;
                int curCol = Col + blurCol;

                if (curRow >= 0 && curRow < h && curCol >= 0 && curCol < w)
                {
                    uchar3 pixel = in[curRow * w + curCol];
                    pixValR += static_cast<int>(pixel.x);
                    pixValG += static_cast<int>(pixel.y);
                    pixValB += static_cast<int>(pixel.z);
                    pixels++;
                }
            }
        }

        // Assign accumulated values to output
        out[Row * w + Col].x = static_cast<unsigned char>(pixValR / pixels);
        out[Row * w + Col].y = static_cast<unsigned char>(pixValG / pixels);
        out[Row * w + Col].z = static_cast<unsigned char>(pixValB / pixels);
    }
}