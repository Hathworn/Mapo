#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void color_to_grey(uchar3 *input_image, uchar3 *output_image, int width, int height)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (col < width && row < height)
    {
        int pos = row * width + col;
        // Calculate the grayscale value once and use it for all three channels
        unsigned char grey_value = static_cast<unsigned char>(input_image[pos].x * 0.2126f + input_image[pos].y * 0.7125f + input_image[pos].z * 0.0722f);
        output_image[pos] = make_uchar3(grey_value, grey_value, grey_value); // Use make_uchar3 for cleaner assignment
    }
}