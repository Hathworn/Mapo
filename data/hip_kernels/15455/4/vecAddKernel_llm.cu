#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAddKernel(float *a, float *b, float *c, int n)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure threads work only within vector bounds
    if (id < n) {
        c[id] = a[id] + b[id];
    }

    // Optimize memory access by using shared memory (if applicable)
    // __shared__ float sharedA[MAX_THREAD];
    // __shared__ float sharedB[MAX_THREAD];
    // sharedA[threadIdx.x] = a[id];
    // sharedB[threadIdx.x] = b[id];
    // __syncthreads();
    // if (id < n) {
    //     c[id] = sharedA[threadIdx.x] + sharedB[threadIdx.x];
    // }
}