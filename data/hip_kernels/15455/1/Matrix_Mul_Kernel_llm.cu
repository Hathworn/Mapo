#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_Mul_Kernel(float* d_M, float* d_N, float* d_P, int Width) {
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float Pvalue = 0;

    for (int ph = 0; ph < Width / TILE_WIDTH; ++ph) {
        // Use coalesced memory access patterns
        if(Row < Width && (ph * TILE_WIDTH + tx) < Width)
            Mds[ty][tx] = d_M[Row * Width + ph * TILE_WIDTH + tx];
        else
            Mds[ty][tx] = 0.0;

        if(Col < Width && (ph * TILE_WIDTH + ty) < Width)
            Nds[ty][tx] = d_N[(ph * TILE_WIDTH + ty) * Width + Col];
        else
            Nds[ty][tx] = 0.0;

        __syncthreads(); // Synchronize threads to ensure shared memory is updated

        for (int k = 0; k < TILE_WIDTH; ++k) {
            Pvalue += Mds[ty][k] * Nds[k][tx];
        }
        __syncthreads(); // Synchronize to prepare for the next tile
    }

    if(Row < Width && Col < Width)
        d_P[Row * Width + Col] = Pvalue;
}