#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Initialize each thread's state with a unique seed
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}