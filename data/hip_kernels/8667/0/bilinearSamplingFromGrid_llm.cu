#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ bool between(int value, int lowerBound, int upperBound) {
    return (value >= lowerBound && value <= upperBound);
}

__device__ void getTopLeft(float x, int width, int& point, float& weight) {
    float xcoord = (x + 1) * (width - 1) / 2;
    point = floor(xcoord);
    weight = 1 - (xcoord - point);
}

__global__ void bilinearSamplingFromGrid(float* inputImages_data, int inputImages_strideBatch, int inputImages_strideChannels, 
                                         int inputImages_strideHeight, int inputImages_strideWidth, float* grids_data, 
                                         int grids_strideBatch, int grids_strideYX, int grids_strideHeight, 
                                         int grids_strideWidth, float* output_data, int output_strideBatch, 
                                         int output_strideChannels, int output_strideHeight, int output_strideWidth, 
                                         int inputImages_channels, int inputImages_height, int inputImages_width, 
                                         int output_width) {
    const int xOut = blockIdx.x * blockDim.y + threadIdx.y;
    if (xOut >= output_width) return;  // Early exit for out of bounds
    const int yOut = blockIdx.y;
    const int width = inputImages_width;
    const int height = inputImages_height;
    const int b = blockIdx.z;

    float yf, xf;
    __shared__ float gridData[32];
    if (threadIdx.y == 0 && threadIdx.x < 32) {
        gridData[threadIdx.x] = grids_data[b * grids_strideBatch + yOut * grids_strideHeight + (blockIdx.x * blockDim.y + threadIdx.x / 2) * grids_strideWidth + threadIdx.x % 2];
    }
    __syncthreads();

    yf = gridData[threadIdx.y * 2];
    xf = gridData[threadIdx.y * 2 + 1];

    int yInTopLeft, xInTopLeft;
    float yWeightTopLeft, xWeightTopLeft;
    getTopLeft(xf, inputImages_width, xInTopLeft, xWeightTopLeft);
    getTopLeft(yf, inputImages_height, yInTopLeft, yWeightTopLeft);

    const int outAddress = output_strideBatch * b + output_strideHeight * yOut + output_strideWidth * xOut;
    const int inTopLeftAddress = inputImages_strideBatch * b + inputImages_strideHeight * yInTopLeft + inputImages_strideWidth * xInTopLeft;
    const int inTopRightAddress = inTopLeftAddress + inputImages_strideWidth;
    const int inBottomLeftAddress = inTopLeftAddress + inputImages_strideHeight;
    const int inBottomRightAddress = inBottomLeftAddress + inputImages_strideWidth;

    float v;
    for(int t = threadIdx.x; t < inputImages_channels; t += blockDim.x) {
        float inTopLeft = (between(xInTopLeft, 0, width-1) && between(yInTopLeft, 0, height-1)) ? inputImages_data[inTopLeftAddress + t] : 0.0f;
        float inTopRight = (between(xInTopLeft+1, 0, width-1) && between(yInTopLeft, 0, height-1)) ? inputImages_data[inTopRightAddress + t] : 0.0f;
        float inBottomLeft = (between(xInTopLeft, 0, width-1) && between(yInTopLeft+1, 0, height-1)) ? inputImages_data[inBottomLeftAddress + t] : 0.0f;
        float inBottomRight = (between(xInTopLeft+1, 0, width-1) && between(yInTopLeft+1, 0, height-1)) ? inputImages_data[inBottomRightAddress + t] : 0.0f;

        v = xWeightTopLeft * yWeightTopLeft * inTopLeft
            + (1 - xWeightTopLeft) * yWeightTopLeft * inTopRight
            + xWeightTopLeft * (1 - yWeightTopLeft) * inBottomLeft
            + (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * inBottomRight;

        output_data[outAddress + t] = v;
    }
}