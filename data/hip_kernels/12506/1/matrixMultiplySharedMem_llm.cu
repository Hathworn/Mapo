#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 40

__global__ void matrixMultiplySharedMem(double * A, double * B, double * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    // Declare shared memory for tiles
    __shared__ double ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ double ds_N[TILE_WIDTH][TILE_WIDTH];

    // Calculate row and column index of C matrix
    int bx = blockIdx.x, by = blockIdx.y,
        tx = threadIdx.x, ty = threadIdx.y,
        Row = by * TILE_WIDTH + ty,
        Col = bx * TILE_WIDTH + tx;

    double Pvalue = 0;

    for (int m = 0; m < (numAColumns + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
        // Load data into shared memory with boundary check
        if (Row < numARows && m * TILE_WIDTH + tx < numAColumns)
            ds_M[ty][tx] = A[Row * numAColumns + m * TILE_WIDTH + tx];
        else
            ds_M[ty][tx] = 0.0;

        if (Col < numBColumns && m * TILE_WIDTH + ty < numBRows)
            ds_N[ty][tx] = B[(m * TILE_WIDTH + ty) * numBColumns + Col];
        else
            ds_N[ty][tx] = 0.0;

        __syncthreads();

        // Perform multiplication for the current tile
        #pragma unroll
        for (int k = 0; k < TILE_WIDTH; ++k) {
            Pvalue += ds_M[ty][k] * ds_N[k][tx];
        }

        __syncthreads();
    }

    // Store result in C if valid index
    if (Row < numCRows && Col < numCColumns)
        C[Row * numCColumns + Col] = Pvalue;
}