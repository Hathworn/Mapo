#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 40

//--------------------------------------------------

// Compute C = A * B

//-------------------------------------------------

__global__ void MatrixMult(int m, int n, int k, double *a, double *b, double *c)
{
    // Optimize block access by calculating thread's row and column
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        double temp = 0.0;
        // Use register to improve memory access speed
        for (int i = 0; i < n; ++i) {
            temp += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = temp;
    }
}