#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved reciprocalKernel with memory coalescing and loop unrolling
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned stride = gridDim.x * blockDim.x;

    // Stride loop to process data in chunks
    for (unsigned i = idx; i < vectorSize; i += stride) {
        data[i] = 1.0 / data[i];
    }
}