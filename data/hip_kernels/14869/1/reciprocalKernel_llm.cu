#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < vectorSize) {
        // Store the reciprocal in a register variable to reduce global memory access
        float reciprocal = 1.0f / data[idx];
        data[idx] = reciprocal;
    }
}