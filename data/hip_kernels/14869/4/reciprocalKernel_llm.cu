#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
    // Compute global thread index
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if the index is within bounds and update the data
    if (idx < vectorSize) {
        data[idx] = __fdiv_rd(1.0f, data[idx]); // Use fast division intrinsic
    }
}