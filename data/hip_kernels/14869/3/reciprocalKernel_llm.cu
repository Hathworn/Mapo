#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory to improve memory access latency
    __shared__ float cache[256];  // Assuming blockDim.x is 256
    if (idx < vectorSize) {
        cache[threadIdx.x] = data[idx];
        __syncthreads();
        data[idx] = 1.0 / cache[threadIdx.x];
    }
}