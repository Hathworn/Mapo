#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory for input data
    extern __shared__ float sharedData[];
    if (idx < vectorSize) {
        sharedData[threadIdx.x] = data[idx];
        __syncthreads();
        // Perform reciprocal operation
        data[idx] = 1.0 / sharedData[threadIdx.x];
    }
}