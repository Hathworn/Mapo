#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure all threads handle one data element
    if (idx < vectorSize) {
        float value = data[idx];
        // Avoid division by zero or invalid operation
        if (value != 0.0f) {
            data[idx] = 1.0f / value;
        } else {
            data[idx] = 0.0f; // Set appropriate value for zero or invalid data
        }
    }
}