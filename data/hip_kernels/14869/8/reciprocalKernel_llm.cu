#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < vectorSize) {
        // Use a local variable to optimize memory access
        float val = data[idx];
        data[idx] = 1.0f / val;
    }
}