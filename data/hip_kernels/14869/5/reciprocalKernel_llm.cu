#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < vectorSize) {
        float val = data[idx];
        // Avoid division if the value is zero to prevent division by zero
        if (val != 0.0) {
            data[idx] = 1.0 / val;
        }
    }
}