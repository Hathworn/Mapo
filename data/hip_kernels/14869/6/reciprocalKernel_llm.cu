#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
    // Calculate global index
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check boundary condition
    if (idx < vectorSize) {
        // Ensure non-zero to avoid division by zero error
        if (data[idx]!= 0.0f) {
            data[idx] = 1.0f / data[idx];
        }
    }
}