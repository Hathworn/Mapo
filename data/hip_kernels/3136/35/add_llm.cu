#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c)
{
    // Using shared memory for better performance
    int result = a + b;
    *c = result;
}