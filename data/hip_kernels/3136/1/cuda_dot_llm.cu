#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_dot(int N, double *a, double *b, double *c)
{
    extern __shared__ double localDot[];
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int localIndex = threadIdx.x;
    
    double localSum = 0;
    for (int i = ix; i < N; i += blockDim.x * gridDim.x) {
        localSum += a[i] * b[i];  // Accumulate dot product contribution
    }

    localDot[localIndex] = localSum;
    __syncthreads();

    for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
        if (localIndex < offset) {
            localDot[localIndex] += localDot[localIndex + offset];  // Reduce within block
        }
        __syncthreads();
    }

    if (localIndex == 0) {
        c[blockIdx.x] = localDot[0];  // Store result from block into the output array
    }
}