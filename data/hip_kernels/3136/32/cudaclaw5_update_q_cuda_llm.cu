#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaclaw5_update_q_cuda(int mbc, double dtdx, double dtdy, double* qold, double* fm, double* fp, double* gm, double* gp)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global x index
    int y = blockIdx.y * blockDim.y + threadIdx.y;  // Calculate global y index
    
    if (x >= mbc && y >= mbc) {  // Boundary check for x and y
        int mq = threadIdx.z;
        int x_stride = blockDim.z;
        int y_stride = (blockDim.x + 2*mbc) * x_stride;
        int i = mq + (x + mbc) * x_stride + (y + mbc) * y_stride;
        qold[i] = qold[i] - dtdx * (fm[i + x_stride] - fp[i])
                           - dtdy * (gm[i + y_stride] - gp[i]);
    }
}