#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaclaw5_compute_cfl(int idir, int mx, int my, int meqn, int mwaves, int mbc, double dx, double dy, double dt, double *speeds, double* cflgrid)
{
    // Calculate grid offset based on x/y direction
    int grid_offset = (idir == 1) ? my : mx;

    // Global thread index for accessing speed and cflgrid
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index is within grid bounds
    if (tid >= grid_offset)
        return;

    double max_cfl = 0.0;
    double dtdx = (idir == 1) ? dt / dx : dt / dy;

    // Optimize loop by processing waves for each thread
    for (int mw = 0; mw < mwaves; ++mw) {
        double s_pos = speeds[mw * grid_offset + tid];
        double s_neg = -s_pos; // Simplified assuming symmetric layout

        // Compare and keep track of maximum CFL
        if (s_pos > 0)
            max_cfl = max(max_cfl, dtdx * s_pos);
        else
            max_cfl = max(max_cfl, dtdx * s_neg);
    }
    
    // Write the maximum CFL value to the grid
    cflgrid[tid] = max_cfl;
}