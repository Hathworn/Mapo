#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_ClassNLLCriterion_updateGradInput_kernel(float *gradInput, float *target, int nframe, int ndim, float grad, int ntarget) {
    // Use blockIdx.x and blockDim.x to distribute work across threads
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < nframe) {
        for (int j = 0; j < ntarget; ++j) {
            int t = static_cast<int>(target[i * ntarget + j]) - 1;
            if (t >= 0) {
                gradInput[i * ndim + t] = grad;
            }
        }
    }
}
```
