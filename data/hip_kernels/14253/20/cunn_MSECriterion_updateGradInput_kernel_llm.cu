#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_MSECriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, float norm, int nframe, int dim)
{
    int k = blockIdx.x;
    float *gradInput_k = gradInput + k * dim;
    float *input_k = input + k * dim;
    float *target_k = target + k * dim;

    int i = threadIdx.x;

    // Use one thread per element if there are enough threads, avoiding unnecessary calculations
    if (i < dim) {
        gradInput_k[i] = norm * (input_k[i] - target_k[i]);
    }
}