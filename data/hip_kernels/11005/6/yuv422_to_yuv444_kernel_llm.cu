#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void yuv422_to_yuv444_kernel(const void * src, void * out, int pix_count) {
    // Calculate thread index
    const int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Return if out of bounds
    if (idx >= pix_count / 2) return;

    uchar4 *src_ptr = (uchar4 *)src + idx * 2;
    uchar4 *dst_ptr = (uchar4 *)out + idx * 3;

    uchar4 pix12 = src_ptr[0];
    uchar4 pix34 = src_ptr[1];

    // Optimize by using a single step assignment
    dst_ptr[0] = {pix12.y, pix12.x, pix12.z, pix12.w}; 
    dst_ptr[1] = {pix12.x, pix12.z, pix34.y, pix34.x}; 
    dst_ptr[2] = {pix34.z, pix34.w, pix34.x, pix34.z}; 
}