#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load value from global memory
    float acc = *accuracy;

    // Perform division using shared memory to reduce redundant global memory access
    __shared__ float sharedAcc;
    if (threadIdx.x == 0) {
        sharedAcc = acc / N;
    }
    __syncthreads();  // Synchronize threads

    // Write result back to global memory
    if (threadIdx.x == 0) {
        *accuracy = sharedAcc;
    }
}