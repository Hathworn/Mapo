#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(float * d_out, float * d_in) {
    // Calculate global thread index using faster multiplication
    unsigned int gid = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Use direct memory access without additional variable
    d_out[gid] = d_in[gid] * d_in[gid];
}