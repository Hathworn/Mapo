#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(float *d_out, const float *d_in) {
    // Use register to store input value for faster access
    const unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;
    const float val = d_in[gid];
    
    // Avoid repeat computation by storing result in a register
    d_out[gid] = val * val;
}