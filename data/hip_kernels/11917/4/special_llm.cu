#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void special(float *d_out, const float * __restrict__ d_in, int size) {
    const unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < size) {
        float x = d_in[gid];
        // Optimize division and powf calculation
        float divisor = x - 2.3f;
        float result = (divisor != 0.0f) ? x / divisor : 0.0f;
        d_out[gid] = result * result * result;
    }
}