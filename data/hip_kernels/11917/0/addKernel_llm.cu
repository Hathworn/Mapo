#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use global thread index for larger arrays
    if (i < N) // Guard against out-of-bounds access assuming size N is defined
    {
        c[i] = a[i] + b[i];
    }
}