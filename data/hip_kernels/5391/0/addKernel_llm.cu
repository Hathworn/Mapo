#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate the global index for the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Perform element-wise addition if within bounds
    if (i < blockDim.x) {
        c[i] = a[i] + b[i];
    }
}