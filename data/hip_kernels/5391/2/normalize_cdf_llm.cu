#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf( unsigned int* d_input_cdf, float* d_output_cdf, int n )
{
    const float normalization_constant = 1.f / d_input_cdf[n - 1]; // Compute normalization constant

    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x; // Compute global index

    if ( global_index_1d < n )
    {
        unsigned int input_value  = d_input_cdf[global_index_1d]; // Load input value
        float output_value = __fmul_rd(input_value, normalization_constant); // Efficient multiplication

        d_output_cdf[global_index_1d] = output_value; // Store output value
    }
}