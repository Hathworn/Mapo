#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tonemap(float* d_x, float* d_y, float* d_log_Y, float* d_cdf_norm, float* d_r_new, float* d_g_new, float* d_b_new, float min_log_Y, float max_log_Y, float log_Y_range, int num_bins, int num_pixels_y, int num_pixels_x)
{
    int ny = num_pixels_y;
    int nx = num_pixels_x;
    int image_index_2d_x = blockIdx.x * blockDim.x + threadIdx.x;
    int image_index_2d_y = blockIdx.y * blockDim.y + threadIdx.y;
    int image_index_1d = nx * image_index_2d_y + image_index_2d_x;
    
    // Check if indices are within image bounds
    if (image_index_2d_x < nx && image_index_2d_y < ny)
    {
        float x = d_x[image_index_1d];
        float y = d_y[image_index_1d];
        float log_Y = d_log_Y[image_index_1d];
        
        // Simplified expression avoiding function calls inside the loop
        float bin_index_norm = (log_Y - min_log_Y) / log_Y_range;
        int bin_index = min(num_bins - 1, int(num_bins * bin_index_norm));
        float Y_new = d_cdf_norm[bin_index];
        
        // Combined repetitive calculations
        float Y_ratio = Y_new / y;
        float X_new = x * Y_ratio;
        float Z_new = (1 - x - y) * Y_ratio;

        // Precomputed constants for linear transformation
        const float r_const0 =  3.2406f, r_const1 = -1.5372f, r_const2 = -0.4986f;
        float r_new = (X_new * r_const0) + (Y_new * r_const1) + (Z_new * r_const2);

        const float g_const0 = -0.9689f, g_const1 =  1.8758f, g_const2 =  0.0415f;
        float g_new = (X_new * g_const0) + (Y_new * g_const1) + (Z_new * g_const2);
        
        const float b_const0 =  0.0557f, b_const1 = -0.2040f, b_const2 =  1.0570f;
        float b_new = (X_new * b_const0) + (Y_new * b_const1) + (Z_new * b_const2);

        // Write results back to global memory
        d_r_new[image_index_1d] = r_new;
        d_g_new[image_index_1d] = g_new;
        d_b_new[image_index_1d] = b_new;
    }
}