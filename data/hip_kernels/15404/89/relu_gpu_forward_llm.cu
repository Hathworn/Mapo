#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for coalesced access
    for (int i = tid; i < N; i += stride) { // Loop with stride
        out[i] = in[i] > 0 ? in[i] : 0;
    }
}