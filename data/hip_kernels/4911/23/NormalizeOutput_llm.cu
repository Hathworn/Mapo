#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Unrolling loop for efficient memory access
    #pragma unroll
    for (; idx < num_elements; idx += stride) {
        int base_idx = idx * 3; // Calculate base index only once
        to_normalize[base_idx] = batch_index;
        to_normalize[base_idx + 1] = class_index;
        to_normalize[base_idx + 2] = static_cast<int64_t>(original[idx]);
    }
}