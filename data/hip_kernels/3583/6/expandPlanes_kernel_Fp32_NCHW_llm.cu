#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expandPlanes_kernel_Fp32_NCHW(float* output, const uint64_t* masks, const float* values, int n) {
    // Improved parallelism by adjusting shared memory loading and mask indexing
    constexpr int kNumThreads = 256;
    constexpr int kNumShmemElements = kNumThreads / 64;

    __shared__ uint64_t shMasks[kNumShmemElements];
    __shared__ float shVals[kNumShmemElements];

    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int planeIndex = index >> 6;

    if (planeIndex >= n) return;

    // Load masks and values for each 64-thread segment efficiently
    int shmemIndex = threadIdx.x >> 6; // Calculate shared memory index
    if (shmemIndex < kNumShmemElements && threadIdx.x % 64 == 0) {
        shMasks[shmemIndex] = masks[planeIndex + shmemIndex];
        shVals[shmemIndex] = values[planeIndex + shmemIndex];
    }
    __syncthreads();

    uint64_t mask = shMasks[shmemIndex];
    float op = (mask & (1ull << (index & 0x3F))) ? shVals[shmemIndex] : 0.0f;

    output[index] = op;
}