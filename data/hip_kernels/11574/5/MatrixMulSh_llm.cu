#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulSh(float *Md, float *Nd, float *Pd, const int WIDTH) {
    // Shared memory tiles for matrices M and N
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // Calculate thread-specific column and row
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

    float Pvalue = 0; // Accumulate the result

    for (int m = 0; m < WIDTH / TILE_WIDTH; m++) {
        // Load tiles into shared memory from global memory
        Mds[threadIdx.y][threadIdx.x] = Md[row * WIDTH + (m * TILE_WIDTH + threadIdx.x)];
        Nds[threadIdx.y][threadIdx.x] = Nd[(m * TILE_WIDTH + threadIdx.y) * WIDTH + col];
        __syncthreads(); // Synchronize threads to ensure all data is loaded

        // Multiply the tiles and accumulate results
        for (int k = 0; k < TILE_WIDTH; k++) {
            Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
        }
        __syncthreads(); // Synchronize threads before loading new tiles
    }

    // Write result to global memory
    Pd[row * WIDTH + col] = Pvalue;
}