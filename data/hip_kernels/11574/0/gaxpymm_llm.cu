#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpymm(double *y, double *a, double *b, int m, int n, int p) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];

    if (bid < m) {
        for (int c = 0; c < p; c++) {
            if (tid < n) {
                // Perform multiplication only if tid < n
                dots_s[tid * p + c] = a[bid * n + tid] * b[tid * p + c];
            }
        }
        __syncthreads();

        // Use a single thread to accumulate values
        if (tid == 0) {
            for (int c = 0; c < p; c++) {
                double sum = 0.0;
                for (int i = 0; i < n; i++) {
                    sum += dots_s[i * p + c];
                }
                y[bid * p + c] = sum;
            }
        }
    }
}