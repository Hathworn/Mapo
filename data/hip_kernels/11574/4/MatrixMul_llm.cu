#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMul(float *Md, float *Nd, float *Pd, const int WIDTH)
{
    // Calculate global thread row and column indices
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < WIDTH && col < WIDTH) // Ensure indices are within bounds
    {
        float Pvalue = 0; // Initialize accumulator
        for (int k = 0; k < WIDTH; ++k)
        {
            // Accumulate results of matrix multiplication
            Pvalue += Md[row * WIDTH + k] * Nd[k * WIDTH + col];
        }
        Pd[row * WIDTH + col] = Pvalue; // Store result
    }
}