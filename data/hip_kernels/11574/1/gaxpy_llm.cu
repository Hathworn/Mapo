#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel function
__global__ void gaxpy(double *y, double *a, double *x, int m, int n) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];

    // Ensure valid block and thread indexing
    if (bid < m && tid < n) {
        dots_s[tid] = a[bid * n + tid] * x[tid];
    }

    __syncthreads();

    // Reduce using parallel reduction (faster accumulation of dots_s)
    for (int stride = n / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            dots_s[tid] += dots_s[tid + stride];
        }
        __syncthreads();
    }

    // Write result to output when thread 0 sums the segment
    if (tid == 0) {
        y[bid] = dots_s[0];
    }
}