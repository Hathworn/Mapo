#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square(int *array, int arrayCount)
{
    // Calculate unique global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce global memory bandwidth
    extern __shared__ int dynamicSmem[];
    if (idx < arrayCount) {
        dynamicSmem[threadIdx.x] = array[idx];
        __syncthreads();
        
        // Perform the computation in shared memory
        dynamicSmem[threadIdx.x] *= dynamicSmem[threadIdx.x];
        
        __syncthreads();
        // Write the result back to global memory
        array[idx] = dynamicSmem[threadIdx.x];
    }
}