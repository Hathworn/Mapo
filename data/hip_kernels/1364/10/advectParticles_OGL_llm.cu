#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void advectParticles_OGL(float2 *part, float2 *v, int dx, int dy, float dt, int lb, size_t pitch) {
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;

    if (gtidx >= dx) return; // Early exit for threads outside domain in X

    float2 pterm, vterm;
    for (int p = 0; p < lb; p++) {
        int fi = gtidy + p;
        if (fi >= dy) break; // Early exit for rows outside domain in Y

        int fj = fi * dx + gtidx;
        pterm = part[fj];

        // Calculate velocity index using faster address computation
        int xvi = __float2int_rd(pterm.x * dx);
        int yvi = __float2int_rd(pterm.y * dy);
        vterm = *((float2*)((char*)v + yvi * pitch) + xvi);

        // Update particle position with improved modulo operation
        pterm.x = fmodf(pterm.x + dt * vterm.x + 1.0f, 1.0f);
        pterm.y = fmodf(pterm.y + dt * vterm.y + 1.0f, 1.0f);

        part[fj] = pterm;
    }
}