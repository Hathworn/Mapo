#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
    // Calculate flattened global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Cache factor value in shared memory to enhance performance
    __shared__ int cached_factor;
    if (threadIdx.x == 0) {
        cached_factor = *factor;
    }
    // Synchronize to ensure all threads have loaded the factor
    __syncthreads();
    
    // Perform iterations with cached factor
    int temp = 0;
    for (int i = 0; i < num_iterations; i++) {
        temp += cached_factor;
    }
    g_data[idx] += temp;
}