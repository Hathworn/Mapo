#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addForces_k(float2 *v, int dx, int dy, int spx, int spy, float fx, float fy, int r, size_t pitch) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    // Move the pointer arithmetic outside the kernel loop to optimize memory access
    float2 *fj = (float2*)((char*)v + (ty + spy) * pitch) + tx + spx;

    // Minimize float2 packing/unpacking by directly operating within registers
    float2 vterm = *fj;

    // Precompute repetitive calculations to reduce instructions
    int tx_offset = tx - r;
    int ty_offset = ty - r;
    float tx2 = tx_offset * tx_offset;
    float ty2 = ty_offset * ty_offset;
    float s = 1.f / (1.f + tx2 * tx2 + ty2 * ty2);

    vterm.x += s * fx;
    vterm.y += s * fy;

    // Store result directly after operations
    *fj = vterm;
}