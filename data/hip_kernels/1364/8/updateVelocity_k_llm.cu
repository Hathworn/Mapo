#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateVelocity_k(float2 *v, float *vx, float *vy, int dx, int pdx, int dy, int lb, size_t pitch) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;

    // Check if the thread is within the domain bounds in X direction
    if (gtidx < dx) {
        // Cache scale value outside the loop for reuse
        float scale = 1.f / (dx * dy);
        
        // Loop through the lb elements per thread in Y direction
        for (int p = 0; p < lb; p++) {
            int fi = gtidy + p;

            // Check if the thread is within the domain bounds in Y direction
            if (fi < dy) {
                int fjr = fi * pdx + gtidx;

                // Load and normalize the velocity terms
                float vxterm = vx[fjr] * scale;
                float vyterm = vy[fjr] * scale;

                // Store the results back into the global memory
                float2 nvterm = {vxterm, vyterm};
                float2 *fj = (float2*)((char*)v + fi * pitch) + gtidx;
                *fj = nvterm;
            }
        }
    }
}