#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diffuseProject_k(float2 *vx, float2 *vy, int dx, int dy, float dt, float visc, int lb) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    // Perform boundary check at the beginning
    if (gtidx >= dx) return;

    for (p = 0; p < lb; p++) {
        int fi = gtidy + p;
        if (fi < dy) {
            int fj = fi * dx + gtidx;
            float2 xterm = vx[fj];
            float2 yterm = vy[fj];

            // Compute the index of the wavenumber
            int iix = gtidx;
            int iiy = (fi > dy / 2) ? (fi - dy) : fi;

            // Velocity diffusion
            float kk = (float)(iix * iix + iiy * iiy); // k^2
            float diff = 1.f / (1.f + visc * dt * kk);
            xterm.x *= diff; xterm.y *= diff;
            yterm.x *= diff; yterm.y *= diff;

            // Velocity projection
            if (kk > 0.f) {
                float rkk = 1.f / kk;
                float rkp = (iix * xterm.x + iiy * yterm.x);
                float ikp = (iix * xterm.y + iiy * yterm.y);
                xterm.x -= rkk * rkp * iix;
                xterm.y -= rkk * ikp * iix;
                yterm.x -= rkk * rkp * iiy;
                yterm.y -= rkk * ikp * iiy;
            }

            vx[fj] = xterm;
            vy[fj] = yterm;
        }
    }
}
```
