#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAdd(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int N)
{
    // Calculate global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Use if (i < N) to prevent accessing out of bounds
    if (i < N) 
    {
        // Perform vector addition
        C[i] = A[i] + B[i];
    }
}