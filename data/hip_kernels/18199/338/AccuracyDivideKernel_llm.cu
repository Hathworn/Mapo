#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel to avoid possible race conditions and improve clarity
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    if (threadIdx.x == 0 && blockIdx.x == 0) { // Ensure single thread executes the division
        *accuracy /= N;
    }
}