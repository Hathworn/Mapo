#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_sparse_to_dense_assign(int n, const unsigned int *idx, float *src, float *trg) {
    // Use shared memory for improved access patterns
    __shared__ unsigned int shared_idx[256];
    __shared__ float shared_src[256];

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load from global to shared memory
    if (id < n) {
        shared_idx[threadIdx.x] = idx[id];
        shared_src[threadIdx.x] = src[shared_idx[threadIdx.x]];
        __syncthreads();
        
        // Copy from shared to global memory
        trg[id] = shared_src[threadIdx.x];
    }
}