#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_dense_to_sparse_block_add(int n, const unsigned *idx, int bsize, float* src, float *trg) {
    // Calculate the global index and the corresponding sparse block index once
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n * bsize) {
        // Use a register to avoid multiple index calculations for the same thread
        int blockIndex = idx[id / bsize];
        int offset = id % bsize;
        // Perform atomic addition with pre-calculated index
        atomicAdd(trg + blockIndex * bsize + offset, src[id]);
    }
}