#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_sparse_to_dense_block_assign_and_multiply(int n, const unsigned *idx, int bsize, float mult, float* src, float *trg) {
    // Use shared memory to optimize memory access
    __shared__ unsigned shared_idx[256];
    
    // Each thread loads one index into shared memory
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    if (tid < bsize) {
        shared_idx[tid] = idx[blockIdx.x * bsize + tid];
    }
    __syncthreads();

    // Calculate the target index in the output array
    int global_index = i * bsize + tid;

    // Check array bounds and perform computation
    if (i < n) {
        trg[global_index] = src[shared_idx[tid] * bsize + tid] * mult;
    }
}