#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_dense_to_sparse_subtract(int n, const unsigned int *idx, float *src, float *trg) {
    // Calculate our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we do not go out of bounds
    if (id < n) {
        unsigned int targetIndex = idx[id]; // Optimize by reducing indirection
        float valueToSubtract = -src[id];   // Precompute subtraction
        atomicAdd(trg + targetIndex, valueToSubtract); // Perform atomic operation
    }
}