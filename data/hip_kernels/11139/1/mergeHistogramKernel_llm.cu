#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TAG_MASK 0xFFFFFFFFU

__global__ void mergeHistogramKernel(uint *d_Histogram, uint *d_PartialHistograms, uint histogramCount) {
    uint sum = 0;

    // Using threadIdx.x to handle multiple bins in a single thread iteration
    for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE) {
        sum += d_PartialHistograms[blockIdx.x * HISTOGRAM_BIN_COUNT + i];
    }

    // Declare shared memory for inter-thread communication
    __shared__ uint data[MERGE_THREADBLOCK_SIZE];
    data[threadIdx.x] = sum;

    // Efficient reduction using shared memory
    for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1) {
        __syncthreads(); // Ensure all partial sums are ready

        if (threadIdx.x < stride) {
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
    }

    // Final result stored by thread zero
    if (threadIdx.x == 0) {
        d_Histogram[blockIdx.x] = data[0];
    }
}