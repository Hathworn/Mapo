#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int is_source_gpu(int i, int j, int radius, int source_active, int src_x, int src_y)
{
    if (!source_active)
        return 0;
    if (sqrt(pow((float)(src_x - i), 2) + pow((float)(src_y - j), 2)) <= radius)
        return 1;
    return 0;
}

__global__ void wireless_src_pulse_kernel(int step, double amp, double MAX_TIME, double TIME_STEP, int radius, int source_active, int src_x, int src_y, double *ua_gpu, double *ub_gpu, double *uc_gpu)
{
    int global_thread_x = blockDim.x * blockIdx.x + threadIdx.x;
    int global_thread_y = blockDim.y * blockIdx.y + threadIdx.y;

    // Calculate the work boundary for each thread
    int i_start = global_thread_x * 32;
    int j_start = global_thread_y * 32;
    int i_final = i_start + 32;
    int j_final = j_start + 32;

    int line_length = gridDim.y * blockDim.y;

    // Check step and apply pulse source or zero out
    if (step < (int)(MAX_TIME / TIME_STEP) / 2) {
        for (int i = i_start; i < i_final; i++) {
            for (int j = j_start; j < j_final; j++) {
                if (is_source_gpu(i, j, radius, 1, src_x, src_y))
                    uc_gpu[i * line_length + j] = amp * fabs(sin(step * M_PI / 4));
            }
        }
    } else if (source_active) {
        for (int i = i_start; i < i_final; i++) {
            for (int j = j_start; j < j_final; j++) {
                if (is_source_gpu(i, j, radius, source_active, src_x, src_y)) {
                    ua_gpu[i * line_length + j] = 0;
                    ub_gpu[i * line_length + j] = 0;
                    uc_gpu[i * line_length + j] = 0;
                }
            }
        }
    }
}