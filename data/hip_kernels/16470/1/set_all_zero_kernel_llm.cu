#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_all_zero_kernel(double *ua_gpu, double *ub_gpu, double *uc_gpu)
{
    // Calculate global index to improve memory access pattern and reduce divergence
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < blockDim.x * gridDim.x) {
        ua_gpu[idx] = 0;
        ub_gpu[idx] = 0;
        uc_gpu[idx] = 0;
    }
}