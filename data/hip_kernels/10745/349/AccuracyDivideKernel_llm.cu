#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Optimize by using the first thread for division
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}