#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate global thread index
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    // Initialize each thread with same seed, unique sequence number
    hiprand_init(seed, tidx, 0, &state[tidx]);
}