#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0; // Use a register for accumulation

    // Unroll comparison and blocking accesses
    while (eidx < numElements) {
        sum += a[eidx] * b[eidx];
        eidx += gridDim.x * DP_BLOCKSIZE;
    }
    shmem[threadIdx.x] = sum; // Store in shared memory

    __syncthreads();

    // Use loop for reduction with warp-synchronous programming
    for (uint stride = DP_BLOCKSIZE / 2; stride > 32; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Warp-level reduction
    if (threadIdx.x < 32) {
        volatile float* mysh = &shmem[threadIdx.x];
        mysh[0] += mysh[32];
        mysh[0] += mysh[16];
        mysh[0] += mysh[8];
        mysh[0] += mysh[4];
        mysh[0] += mysh[2];
        mysh[0] += mysh[1];
    }

    // Store result for this block
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}