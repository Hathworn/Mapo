#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Check if the most probable label is the true label
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            // Iterate through numOut to count how many have max probability
            for (int i = 0; i < numOut; i++) {
                numMax += __ldg(&probs[i * numCases + tx]) == maxp;
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}