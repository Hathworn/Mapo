#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normCalc(float *d_A, float *d_B, int n) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < n) {
        // Declare local variables to avoid shared memory bank conflicts
        float mu = 0.0f;
        float sigma = 0.0f;
        
        // Calculate mean (mu)
        for (int row = 0; row < n; row++){
            mu += d_A[col*n+row];
        }
        mu /= n;

        // Calculate standard deviation (sigma)
        for (int row = 0; row < n; row++){
            float diff = d_A[col*n+row] - mu;
            sigma += diff * diff; // Replaced powf with multiplication for efficiency
        }
        sigma = sqrt(sigma / n);

        // Normalize values
        for (int row = 0; row < n; row++) {
            if (sigma == 0.0f) {
                d_B[row*n+col] = 0.0f;
            } else {
                d_B[row*n+col] = (d_A[col*n+row] - mu) / sigma;
            }
        }
    }
}