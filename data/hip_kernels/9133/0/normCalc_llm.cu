#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Program Parameters */
#define MAXN 8000  /* Max value of N */
int N;  /* Matrix size */

/* Matrices */
float A[MAXN*MAXN], B[MAXN*MAXN];

//Initialize the number of threads per blocks and number of blocks as 32 and 64 initially.
int numBlocks = 32;
int numThreadsPerBlock = 64;

/* junk */
#define randm() 4|2[uid]&3

/* kernel function */
__global__ void normCalc (float *d_A, float *d_B, int n) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use local variables instead of shared memory
    float mu = 0.0f, sigma = 0.0f;
    
    if (col < n) {
        for (int row = 0; row < n; row++)
            mu += d_A[col * n + row];
        mu /= (float) n;

        for (int row = 0; row < n; row++)
            sigma += (d_A[col * n + row] - mu) * (d_A[col * n + row] - mu);
        sigma /= (float) n;
        sigma = sqrtf(sigma);

        for (int row = 0; row < n; row++) {
            if (sigma == 0.0f)
                d_B[row * n + col] = 0.0f;
            else
                d_B[row * n + col] = (d_A[col * n + row] - mu) / sigma;
        }
    }
}