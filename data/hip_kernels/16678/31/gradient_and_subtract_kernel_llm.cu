#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gradient_and_subtract_kernel(float *in, float *grad_x, float *grad_y, float *grad_z)
{
    // Calculate flattened global index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Boundary check
    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z) return;

    // Precompute index multiplier
    long int xStride = c_Size.x;
    long int yStride = c_Size.y * xStride;
    
    // Calculate linear index for 3D array
    long int id = k * yStride + j * xStride + i;

    // Calculate neighboring indices only once
    long int id_x = id + 1;
    long int id_y = id + xStride;
    long int id_z = id + yStride;

    // Compute gradients and subtract based on boundary conditions
    if (i < c_Size.x - 1)
        grad_x[id] -= (in[id_x] - in[id]) / c_Spacing.x;
    if (j < c_Size.y - 1)
        grad_y[id] -= (in[id_y] - in[id]) / c_Spacing.y;
    if (k < c_Size.z - 1)
        grad_z[id] -= (in[id_z] - in[id]) / c_Spacing.z;
}