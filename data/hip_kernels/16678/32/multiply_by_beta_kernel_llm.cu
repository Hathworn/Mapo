#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiply_by_beta_kernel(float * input, float * output, float beta)
{
    // Calculate global thread index in a 1D flattened space for better memory access
    unsigned int id = (blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x) * blockDim.z * blockDim.y * blockDim.x
                      + (threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x);

    // Check against total size instead of individual dimensions
    long int total_size = c_Size.x * c_Size.y * c_Size.z;
    if (id >= total_size) return;

    // Update output using computed linear index
    output[id] = input[id] * beta;
}