#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function using precomputed flattened index
__global__ void set_volume_to_constant(float *out, float value) 
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < c_Size.x && j < c_Size.y && k < c_Size.z) {
        long int id = ((k * c_Size.y + j) * c_Size.x + i);
        out[id] = value;
    }
}