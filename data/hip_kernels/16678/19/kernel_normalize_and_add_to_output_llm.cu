#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_normalize_and_add_to_output(float * dev_vol_in, float * dev_vol_out, float * dev_accumulate_weights, float * dev_accumulate_values)
{
    // Optimize index calculations using built-in functions
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Boundary check to exit threads not processing valid data
    if (i >= c_volSize.x || j >= c_volSize.y || k >= c_volSize.z) return;

    // Calculate the index more efficiently
    long int out_idx = i + ((j + k * c_volSize.y) * c_volSize.x);

    float eps = 1e-6;

    // Use a single conditional for normalization check
    if (c_normalize && abs(dev_accumulate_weights[out_idx]) > eps) {
        dev_vol_out[out_idx] = dev_vol_in[out_idx] + (dev_accumulate_values[out_idx] / dev_accumulate_weights[out_idx]);
    } else {
        dev_vol_out[out_idx] = dev_vol_in[out_idx] + (c_normalize ? 0 : dev_accumulate_values[out_idx]);
    }
}