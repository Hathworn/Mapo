#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtract_kernel(float * in1, float * in2, float * out)
{
    // Calculate the global index of each thread using only one calculation
    unsigned long int id = (blockIdx.z * blockDim.z + threadIdx.z) * c_Size.y * c_Size.x +
                           (blockIdx.y * blockDim.y + threadIdx.y) * c_Size.x +
                           (blockIdx.x * blockDim.x + threadIdx.x);

    // Check if index is within bounds
    if (blockIdx.x * blockDim.x + threadIdx.x < c_Size.x && 
        blockIdx.y * blockDim.y + threadIdx.y < c_Size.y && 
        blockIdx.z * blockDim.z + threadIdx.z < c_Size.z) 
    {
        // Perform the subtraction operation
        out[id] = in1[id] - in2[id];
    }
}