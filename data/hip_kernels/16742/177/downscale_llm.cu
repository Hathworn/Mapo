#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Function for calculating the inverse translation index with offset and scaling
__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

// Function for calculating the translation index with scaling
__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

// Inline function for index calculation
__device__ __forceinline__ size_t idx(const size_t nc, const size_t height, const size_t width, const size_t y, const size_t x) {
    return (nc * height + y) * width + x;
}

// Optimized kernel function for downscaling
__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    // Calculate global thread ID
    long ii = blockIdx.y * blockDim.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
    
    // Boundary check
    if (ii >= no_elements) return;
    
    // Loop through scale factors and update gradInput_data
    #pragma unroll  // Unroll loop for better performance
    for (int i = 0; i < scale_factor; i++) {
        for (int j = 0; j < scale_factor; j++) {
            int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
            gradInput_data[ii] += gradOutput_data[ipidx];
        }
    }
}