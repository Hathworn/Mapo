#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel function
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use atomic operation to safely update shared accuracy variable across threads
    atomicExch(accuracy, *accuracy / N);
}