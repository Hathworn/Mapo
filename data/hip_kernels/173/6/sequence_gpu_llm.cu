#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sequence_gpu(int *d_ptr, int length)
{
    // Use shared memory to minimize global memory access
    __shared__ int shared_mem[256]; // Assume blockDim.x <= 256

    int elemID = blockIdx.x * blockDim.x + threadIdx.x;

    if (elemID < length)
    {
        shared_mem[threadIdx.x] = elemID; // Load data into shared memory
        __syncthreads(); // Synchronize threads within the block
        d_ptr[elemID] = shared_mem[threadIdx.x]; // Store data from shared memory to global memory
    }
}