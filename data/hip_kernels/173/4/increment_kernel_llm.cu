#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void increment_kernel(int *g_data, int inc_value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) // Boundary check added
    {
        g_data[idx] += inc_value; // Simplified increment operation
    }
}