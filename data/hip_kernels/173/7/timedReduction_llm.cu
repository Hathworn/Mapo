#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void timedReduction(const float *input, float *output, clock_t *timer)
{
    extern __shared__ float shared[];

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int idx = tid + 2 * bid * blockDim.x;

    if (tid == 0) timer[bid] = clock();

    // Copy input using stride to improve memory access pattern
    shared[tid] = input[idx];
    shared[tid + blockDim.x] = input[idx + blockDim.x];

    // Perform reduction to find minimum with improved stride
    for (int d = blockDim.x; d > 0; d >>= 1) // Use bit shift for division by 2
    {
        __syncthreads();

        if (tid < d)
        {
            float f0 = shared[tid];
            float f1 = shared[tid + d];

            // Min operation in a single statement
            shared[tid] = fminf(f0, f1);
        }
    }

    // Write result.
    if (tid == 0) output[bid] = shared[0];

    __syncthreads();

    if (tid == 0) timer[bid+gridDim.x] = clock();
}