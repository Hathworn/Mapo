#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    // Calculate global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure to use only threads corresponding to valid indices
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}