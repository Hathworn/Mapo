#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Check that both offset and filter are in bounds
    if (offset < size && filter < n) {
        // Compute the linear index and add bias
        output[(batch * n + filter) * size + offset] += biases[filter];
    }
}