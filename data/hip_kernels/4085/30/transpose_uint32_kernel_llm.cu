#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_uint32_kernel(uint32_t *src, uint32_t *dst, int src_h, int src_w, int src_align, int dst_align)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute (i, j) for current thread
    int j = index / src_h;  // Calculate column index
    int i = index % src_h;  // Calculate row index

    // Ensure within bounds
    if (j < src_w)
    {
        // Perform transposition
        dst[j * dst_align / 32 + i] = src[i * src_align + j];
    }
}