#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    // Calculate global thread ID
    int idx = blockDim.x * (blockDim.y * blockIdx.z + blockIdx.y) + threadIdx.x;
    
    if (idx < n * size) {
        int filter = idx / size;
        int offset = idx % size;
        int batch = idx / (n * size);

        // Perform scaling operation
        output[idx] *= biases[filter];
    }
}