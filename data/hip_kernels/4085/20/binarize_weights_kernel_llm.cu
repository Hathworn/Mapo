#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Utilize shared memory to reduce global memory accesses
    extern __shared__ float shared_weights[];
    float mean = 0;

    // Load weights into shared memory
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[i] = weights[f*size + i];
    }
    __syncthreads();

    // Calculate mean using shared memory
    for (int i = 0; i < size; ++i) {
        mean += fabs(shared_weights[i]);
    }
    mean /= size;

    // Assign binary weights using shared memory
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f*size + i] = (shared_weights[i] > 0) ? mean : -mean;
    }
}