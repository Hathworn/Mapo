#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized to use shared memory for the largest element - reduces global memory accesses
__device__ void softmax_device(int n, float *input, float temp, float *output) {
    extern __shared__ float shared[];
    float *largest_ptr = shared;
    float sum = 0.0f;
    
    // Find the largest element using shared memory - helps with thread divergence
    float largest = -INFINITY;
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        largest = fmaxf(largest, input[i]);
    }
    
    *largest_ptr = largest;
    __syncthreads();
    
    // Adjust input values and compute the normalization sum
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        float e = expf((input[i] - *largest_ptr) / temp);
        atomicAdd(&sum, e); // thread-safe accumulation
        output[i] = e;
    }

    __syncthreads();
    
    // Normalize the output using the sum
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        output[i] /= sum;
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output) {
    int b = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (b >= batch) return;
    softmax_device(n, input + b * offset, temp, output + b * offset);
}