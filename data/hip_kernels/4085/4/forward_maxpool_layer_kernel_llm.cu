#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int c = in_c;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int out_index = id;
    int b = id / (w * h * c);
    int k = (id / (w * h)) % c;
    int i = (id / w) % h;
    int j = id % w;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    float max = -INFINITY;
    int max_i = -1;

    // Optimize by using a single loop for iterating through the window
    for (int l = 0; l < size * size; ++l) {
        int x_offset = l % size;
        int y_offset = l / size;
        int cur_h = h_offset + i * stride + y_offset;
        int cur_w = w_offset + j * stride + x_offset;
        if (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w) {
            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            float val = input[index];
            if (val > max) {
                max = val;
                max_i = index;
            }
        }
    }
    output[out_index] = max;
    indexes[out_index] = max_i;
}