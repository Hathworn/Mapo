#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_int8_to_f32(int8_t* input_int8, size_t size, float *output_f32, float multipler)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for grid-stride loop

    // Use grid-stride loop to improve performance and handle larger arrays
    for (int i = idx; i < size; i += stride) {
        output_f32[i] = input_int8[i] * multipler;
    }
}