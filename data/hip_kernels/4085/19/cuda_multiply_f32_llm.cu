#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_multiply_f32(float *input_output, size_t size, float multipler)
{
    // Use shared memory for faster access
    extern __shared__ float shared_input_output[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (idx < size) {
        shared_input_output[threadIdx.x] = input_output[idx];
        __syncthreads(); // Ensure all threads have loaded their data into shared memory

        // Perform multiplication
        shared_input_output[threadIdx.x] *= multipler;
        
        __syncthreads(); // Ensure all threads have completed processing

        // Write back to global memory
        input_output[idx] = shared_input_output[threadIdx.x];
    }
}