#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    // Efficient computation of the global index
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    
    // Use extended Euclidean method to determine indices, minimizing divisions
    size_t id = i;
    int out_w = id % (w * stride);
    id /= (w * stride);
    int out_h = id % (h * stride);
    id /= (h * stride);
    int out_c = id % c;
    int b = id / c;
    
    int in_w = out_w / stride;
    int in_h = out_h / stride;
    
    int in_index = b * w * h * c + out_c * w * h + in_h * w + in_w;
    
    // Conditional logic remains unchanged, but computation ordered for better readability and alignment
    if (forward) {
        out[i] += scale * x[in_index];
    } else {
        atomicAdd(x + in_index, scale * out[i]);
    }
}