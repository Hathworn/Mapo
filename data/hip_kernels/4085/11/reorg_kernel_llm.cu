#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure the thread index is within the array bounds
    if (i >= N) return;

    // Calculate indexes for input and output locations
    const int in_index = i;
    const int in_w = i % w;
    i /= w;
    const int in_h = i % h;
    i /= h;
    const int in_c = i % c;
    i /= c;
    const int b = i % batch;

    const int out_c = c / (stride * stride);

    const int c2 = in_c % out_c;
    const int offset = in_c / out_c;
    const int w2 = in_w * stride + offset % stride;
    const int h2 = in_h * stride + offset / stride;

    // Calculate output index using precomputed values
    const int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    // Assign the output using the calculated index
    out[in_index] = x[out_index];
}