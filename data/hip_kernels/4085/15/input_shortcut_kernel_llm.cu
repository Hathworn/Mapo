#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void input_shortcut_kernel(float *in, int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index computation
    int totalThreads = gridDim.x * blockDim.x; // Total threads available

    for (int id = index; id < size; id += totalThreads) { // Loop to allow multiple iterations
        int i = id % minw;
        int j = (id / minw) % minh; // Direct division for j 
        int k = (id / (minw * minh)) % minc; // Direct division for k
        int b = id / (minw * minh * minc); // Direct division for b

        int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
        int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));

        out[out_index] = in[out_index] + add[add_index];
    }
}