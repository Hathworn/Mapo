#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_input_shortcut_kernel(float *in, int size, float *add, float *out)
{
    // Using a combined thread index to improve code readability
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Utilizing loop unrolling to reduce branch divergence
    for (int i = id; i < size; i += gridDim.x * blockDim.x) {
        out[i] = in[i] + add[i];
    }
}