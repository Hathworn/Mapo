#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation.
    if (i >= N) return;

    int total_layers_spatial = layers * spatial;
    int in_s = i % spatial;
    int temp = i / spatial; // Reuse temporary variable to reduce calculations.
    int in_c = temp % layers;
    int b = temp / layers;

    int i1 = b * total_layers_spatial + in_c * spatial + in_s;
    int i2 = b * total_layers_spatial + in_s * layers + in_c;

    out[forward ? i2 : i1] = x[forward ? i1 : i2]; // Use conditional statement to reduce branching.
}