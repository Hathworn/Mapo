#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    // Use blockIdx.x + blockIdx.y * gridDim.x for 2D block indices
    int index = blockIdx.x * blockDim.x + threadIdx.x; 
    int gridStride = gridDim.x * blockDim.x; // Calculate grid stride

    for (int i = index; i < N; i += gridStride) // Loop to cover all elements
    {
        int f = (i / spatial) % filters; // Calculate filter index
        x[i] = (x[i] - mean[f]) / (sqrtf(variance[f]) + .000001f); // Normalize
    }
}