#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use blockIdx.x for better grid utilization
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        // Optimize memory access pattern
        X[i * INCX] = ALPHA;
    }
}