#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;  // Calculate stride for all threads
    for (int i = tid; i < N; i += stride) {  // Loop with stride to process all elements
        out[i] = in[i] > 0 ? in[i] : 0;  // Simplified relu operation
    }
}