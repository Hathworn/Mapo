#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
const int DIMBLOCKX=32;

//DEVICE

//HOST
__global__ void kernelSum_Column_Matrix(float* matrix, float* array, int tam){
    __shared__ float shareMatrix[DIMBLOCKX];

    float value = 0;
    int col = blockIdx.x;
    int step = tam / blockDim.x;
    int posIni = col * tam + threadIdx.x * step;

    // Optimize loop by unrolling
    for(int i = 0; i < step; i += 4){
        value += matrix[posIni + i];
        if(i + 1 < step) value += matrix[posIni + i + 1];
        if(i + 2 < step) value += matrix[posIni + i + 2];
        if(i + 3 < step) value += matrix[posIni + i + 3];
    }

    shareMatrix[threadIdx.x] = value;
    __syncthreads();

    // Use warp reduction for summing values in shared memory
    if(threadIdx.x < 16) {
        shareMatrix[threadIdx.x] += shareMatrix[threadIdx.x + 16];
    }
    __syncthreads();
    if(threadIdx.x < 8) {
        shareMatrix[threadIdx.x] += shareMatrix[threadIdx.x + 8];
    }
    __syncthreads();
    if(threadIdx.x < 4) {
        shareMatrix[threadIdx.x] += shareMatrix[threadIdx.x + 4];
    }
    __syncthreads();
    if(threadIdx.x < 2) {
        shareMatrix[threadIdx.x] += shareMatrix[threadIdx.x + 2];
    }
    __syncthreads();
    if(threadIdx.x == 0) {
        array[blockIdx.x] = shareMatrix[0] + shareMatrix[1];
    }
}