#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelVector_x_constant(float* arr, int n, int k)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index is within bounds
    if (idx < n)
    {
        // Perform multiplication by constant
        arr[idx] *= k; // Optimize by using shorthand operator
    }
}