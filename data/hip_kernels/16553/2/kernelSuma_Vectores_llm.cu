#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelSuma_Vectores(float* array_A, float* array_B, int _size) {
    // Calculate global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index is within bounds for arrays
    if (idx < _size) {
        // Perform vector addition
        array_A[idx] += array_B[idx];
    }
}