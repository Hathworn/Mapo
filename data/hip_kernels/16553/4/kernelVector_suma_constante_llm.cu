#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelVector_suma_constante(float* __restrict__ array, int _size, int _constant){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Process multiple elements per thread to improve memory coalescing
    for(int i = idx; i < _size; i += gridDim.x * blockDim.x) {
        array[i] += _constant;
    }
}