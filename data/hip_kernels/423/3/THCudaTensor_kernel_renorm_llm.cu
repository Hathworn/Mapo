#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_renorm(float *data, const float value, const long size, const float maxnorm)
{
    __shared__ float buffer[32];
    long tx = threadIdx.x;
    long bx = blockIdx.x;
    long step = blockDim.x;
    float *row = data + size * bx;

    buffer[tx] = 0;

    // Calculate norm for a block
    for (long i = tx; i < size; i += step)
    {
        buffer[tx] += powf(fabsf(row[i]), value);
    }
    
    // Reduce within the block
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
    {
        __syncthreads();
        if (tx < stride)
        {
            buffer[tx] += buffer[tx + stride];
        }
    }

    // Normalize if necessary
    if (tx == 0)
    {
        float norm = powf(buffer[0], 1.0f / value);
        if (norm > maxnorm)
        {
            norm = maxnorm / (norm + 1e-7f);
            buffer[0] = norm;
        }
        else
        {
            buffer[0] = 1.0f;  // No need to renormalize
        }
    }

    __syncthreads();
    // Renormalize rows if needed
    for (long i = tx; i < size; i += step)
    {
        row[i] *= buffer[0];
    }
}