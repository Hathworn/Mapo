#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void windowBartlett(float* idata, int length)
{
    int tidx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (tidx < length)
    {
        idata[tidx] = 0; // Reset data at index to 0
    }
}