#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void windowHamming2d(float* idata, int length, int height)
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    
    if (tidx < length && tidy < height)
    {
        // Precalculate values to avoid redundant calculations
        float tidy_term = 0.54 - 0.46 * cosf(2 * tidy * PI_F / (height - 1));
        float tidx_term = 0.54 - 0.46 * cosf(2 * tidx * PI_F / (length - 1));
        
        // Store the computed result in the idata array 
        idata[tidy * length + tidx] = tidy_term * tidx_term;
    }
}