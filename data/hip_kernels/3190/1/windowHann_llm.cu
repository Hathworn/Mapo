#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void windowHann(float* idata, int length)
{
    // Optimize with precomputed common terms and ensure alignment
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;

    if (tidx < length)
    {
        float factor = 2 * PI_F / (length - 1); // Precompute the constant factor
        idata[tidx] = 0.5 * (1 + cos(tidx * factor)); // Use the precomputed factor
    }
}