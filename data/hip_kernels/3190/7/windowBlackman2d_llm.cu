#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void windowBlackman2d(float* idata, int length, int height)
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;

    // Check if the thread is within bounds
    if (tidx < length && tidy < height)
    {
        // Pre-calculate factors for performance
        float factor_tidy = 2 * PI_F * tidy / (height - 1);
        float factor_tidx = 2 * PI_F * tidx / (length - 1);

        // Calculate Blackman window coefficients
        float cos_term_tidy = cos(factor_tidy);
        float sin_term_tidy = sin(2 * factor_tidy);
        float cos_term_tidx = cos(factor_tidx);
        float sin_term_tidx = sin(2 * factor_tidx);

        // Apply the 2D window function
        idata[tidy * length + tidx] = (0.74 / 2 * -0.5 * cos_term_tidy + 0.16 / 2 * sin_term_tidy)
                                    * (0.74 / 2 * -0.5 * cos_term_tidx + 0.16 / 2 * sin_term_tidx);
    }
}