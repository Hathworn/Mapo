#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void windowHamming(float* idata, int length)
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    int totalThreads = gridDim.x * blockDim.x;

    // Unroll the loop to improve memory access and execution efficiency
    for (; tidx < length; tidx += totalThreads)
    {
        idata[tidx] = 0.54f - 0.46f * cosf(2.0f * tidx * PI_F / (length - 1));
    }
}