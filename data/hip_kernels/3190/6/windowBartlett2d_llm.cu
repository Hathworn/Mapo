#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void windowBartlett2d(float* idata, int length, int height)
{
    // Calculate global 1D index for simplicity
    int global_index = threadIdx.y * blockDim.x + threadIdx.x + 
                       blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * length;

    // Ensure index is within bounds of the matrix dimensions
    if (global_index < length * height)
    {
        idata[global_index] = 0;
    }
}