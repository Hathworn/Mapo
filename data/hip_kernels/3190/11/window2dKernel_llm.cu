#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void window2dKernel(float* __restrict__ idata, const float* __restrict__ window, int width, int height)
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    if (tidx < width && tidy < height)
    {
        int index = tidy * width + tidx; // Compute index once and reuse
        idata[index] = window[index] * idata[index];
    }
}