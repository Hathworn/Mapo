#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void windowKernel(float* idata, float* window, int width, int height)
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    int tidy = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Use a single index to reduce arithmetic operations
    int index = tidy * width + tidx;

    // Check once and reduce operations, using if statement only for valid indices
    if (tidx < width && tidy < height)
    {
        idata[index] *= window[tidx];
    }
}