#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use constant memory for PI_F
__constant__ float c_PI_F = 3.14159265358979323846;

__global__ void windowHann2d(float* idata, int length, int height)
{
    // Calculate proper indices and stride
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y;
    // Ensure valid indices within bounds
    if (tidx < length && tidy < height)
    {
        // Precompute constant terms for efficiency
        float cosTermY = cos(2 * tidy * c_PI_F / (height - 1));
        float cosTermX = cos(2 * tidx * c_PI_F / (length - 1));
        // Use registers for repeated calculations to reduce memory access
        float hannValueY = 0.5f * (1 + cosTermY);
        float hannValueX = 0.5f * (1 + cosTermX);
        // Calculate final Hann window value
        idata[tidy * length + tidx] = hannValueY * hannValueX;
    }
}