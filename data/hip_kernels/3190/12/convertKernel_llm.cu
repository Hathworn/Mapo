#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertKernel(const short* __restrict__ idata, float* __restrict__ odata, int size)
{
    int tidx = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation order
    if(tidx < size) 
        odata[tidx] = __half2float(idata[tidx]); // Use half-precision conversion when applicable
}