#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void zeroFillingKernel(float* idata, int row, int length, int height)
{
    // Calculate the unique thread index within the grid
    int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    int tidy = blockIdx.y * blockDim.y + threadIdx.y; // Added threadIdx.y for 2D

    // Check if the thread is within bounds
    if(tidx < length && tidy < height)
    {
        // Index calculation performed once to minimize arithmetic operations
        int index1 = tidx + (row + tidy) * length;
        int index2 = tidx + (row - tidy) * length;

        // Set data to zero using precomputed indices
        idata[index1] = 0;
        if (tidy != 0) idata[index2] = 0; // Conditional to prevent redundant write
    }
}