#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeGlobalKernel(float* idata, float* odata, int width, int height)
{
    // Calculate the global row and column indexes for input and output matrices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure indices are within bounds before transposing elements
    if (x < width && y < height)
    {
        odata[x * height + y] = idata[y * width + x];
    }
}