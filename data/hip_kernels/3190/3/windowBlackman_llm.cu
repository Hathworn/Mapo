#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void windowBlackman(float* idata, int length)
{
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tidx < length)
    {
        // Precompute reused expressions to enhance performance
        float factor = static_cast<float>(tidx) / (length - 1);
        float cosTerm = cos(2 * PI_F * factor);
        float sinTerm = sin(4 * PI_F * factor);
        
        // Compute windowed value using precomputed terms
        idata[tidx] = 0.37 * cosTerm + 0.08 * sinTerm;
    }
}