#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

int answersNumber;
int categoriesNumber;
int atribsNumber;

__global__ void searchWithCuda(double *resultPossibilities, char *query, char *atribsValues, double *possibilities, int *queryPrefix, int *atribsPrefix, int *answersNumber, int *categoriesNumber, int *atribsNumber)
{
    int category_id = blockIdx.x;  // categories
    int atrib_id = blockIdx.y;     // atribs

    // Find the start of the attribute string in query and atribsValues
    char *queryAtrib = query + queryPrefix[category_id];
    int queryAtribLength = queryPrefix[category_id + 1] - queryPrefix[category_id];

    char *currAtrib = atribsValues + atribsPrefix[atrib_id];
    int currAtribLength = atribsPrefix[atrib_id + 1] - atribsPrefix[atrib_id];

    if (queryAtribLength == currAtribLength)
    {
        // Optimize comparison by using a single loop and return immediately if not matched
        bool equal = true;
        for (int i = 0; i < queryAtribLength; ++i)
        {
            if (queryAtrib[i] != currAtrib[i])
            {
                equal = false;
                break;
            }
        }
        if (equal)
        {
            // Optimize the memory access pattern
            int resultIndex = *categoriesNumber * blockDim.y * i + category_id;
            int possibilitiesIndex = *atribsNumber * i + atrib_id;
            for (int i = 0; i < *answersNumber; ++i)
            {
                resultPossibilities[resultIndex] = possibilities[possibilitiesIndex];
            }
        }
    }
}