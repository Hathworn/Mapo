#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll if condition to potentially reduce warp divergence
    if (tid < N) {
        // Optimize with a conditional assignment
        ingrad[tid] = indata[tid] > 0 ? outgrad[tid] : 0.0f;
    }
}