#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ __forceinline__ size_t idx(const size_t nc, const size_t height, const size_t width, const size_t y, const size_t x) {
    return (nc * height + y) * width + x;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x;
    ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
    
    if (ii >= no_elements) return;

    // Precompute the division results to reduce redundant computation
    int d2_scaled = d2 * scale_factor;
    int d3_scaled = d3 * scale_factor;

    int x = ii / ((d1 * d2_scaled) * d3_scaled);
    int y = (ii / (d2_scaled * d3_scaled)) % d1;
    int z = (ii / d3_scaled) % d2_scaled;
    int w = ii % d3_scaled;

    // Use shared memory for accumulation
    float accumulator = 0;

    for (int i = 0; i < scale_factor; i++) {
        for (int j = 0; j < scale_factor; j++) {
            int ipidx = (((x * d1 + y) * d2) + (z / scale_factor)) * d3 + (w / scale_factor);
            ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
            accumulator += gradOutput_data[ipidx];
        }
    }

    gradInput_data[ii] = accumulator;
}