#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void conv(const float* gm, float* convolved, int bh, int bw, int ih, int iw, int ch, int cw, int smH, int smW, int k, float* sm, int gID, int tID, int nT, int rel_row, int rel_col, int nRows, int stopPrefetchRowID, int lastActiveThreadID) {
    for(int i=k; i<=nRows; i++) {
        // Use register storage to reduce memory access latency
        float reg;
        float regArr[K];
        if(i <= stopPrefetchRowID) {
            reg = gm[i * iw + gID];
            if(tID == lastActiveThreadID) {
                for(int j=1; j<=k-1; j++) {
                    regArr[j] = gm[(i * iw) + gID + j];
                }
            }
        }
        
        // Optimize loop indexing with precomputed offsets
        float imgPixels[K*K];
        int offset = i * smW + tID;
        for(int r=i-k; r<i; r++) {
            for(int c=0; c<k; c++) {
                imgPixels[(r-i+k)*k + c] = sm[(r * smW) + tID + c];
            }
        }
        
        // Use shared memory for computation to increase speed
        float convolvedCell = 0.0;
        for(int c=0; c<k*k; c++) {
            convolvedCell += cm[c]*imgPixels[c];
        }

        // Avoid unnecessary conditional checks and do boundary check
        int cID = ((rel_row * bh + (i-k)) * cw) + (rel_col * nT) + tID;
        if(cID >= 0 && cID < ch*cw) {
            convolved[cID] = convolvedCell;
        }

        __syncthreads();

        if(i <= stopPrefetchRowID) {
            sm[offset] = reg;
            if(tID == lastActiveThreadID) {
                for(int j=1; j<=k-1; j++) {
                    sm[offset + j] = regArr[j];
                }
            }
        }

        __syncthreads();
    }
}

__global__ void conv_kernel(const float* gm, float* convolved, int bh, int bw, int ih, int iw, int ch, int cw, int smH, int smW, int k) {
    int tID = threadIdx.x;
    int bID = blockIdx.x;
    int nT = blockDim.x;
    int nB = gridDim.x;
    int nBx = iw / nT;

    bool isBottomBorder = (bID >= nB - nBx);
    bool isRightBorder = ((bID+1) % nBx == 0);

    __shared__ float sm[ (BLCH + K - 1) * (BLCW + K - 1) ];

    int rel_row = bID / nBx;
    int rel_col = bID % nBx;
    int gID = (rel_row * bh * iw) + (rel_col * nT) + tID;

    for(int i=0; i<k; i++) {
        int sID = i * smW + tID;
        sm[sID] = gm[i * iw + gID];
        if(!isRightBorder && tID == nT-1) {
            for(int j=1; j<=k-1; j++) {
                sID = (i * smW) + tID + j;
                sm[sID] = gm[i * iw + gID + j];
            }
        }
    }

    __syncthreads();

    int nRows, stopPrefetchRowID, lastActiveThreadID;
    if(!isBottomBorder && !isRightBorder) {
        lastActiveThreadID = nT - 1;
        nRows = bh + k - 1;
        stopPrefetchRowID = nRows;
    } else if(isBottomBorder && isRightBorder) {
        if(tID < (nT - (k-1))) {
            nRows = bh;
            stopPrefetchRowID = nRows - 1;
            lastActiveThreadID = nT - k;
        } else return;
    } else {
        if(isBottomBorder) {
            nRows = bh;
            stopPrefetchRowID = nRows - 1;
            lastActiveThreadID = nT - 1;
        } else if(isRightBorder) {
            if(tID < (nT - (k-1))) {
                nRows = bh + k - 1;
                stopPrefetchRowID = nRows;
                lastActiveThreadID = nT - k;
            } else return;
        }
    }

    conv(gm, convolved, bh, bw, ih, iw, ch, cw, smH, smW, k, sm, gID, tID, nT, rel_row, rel_col, nRows, stopPrefetchRowID, lastActiveThreadID);
}