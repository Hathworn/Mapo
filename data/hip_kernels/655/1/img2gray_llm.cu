#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput) {
    // Use shared memory to reduce global memory access
    __shared__ unsigned char sharedInput[1024 * Channels]; // Adjust size according to block size and Channels
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int threadId = threadIdx.y * blockDim.x + threadIdx.x;

    if (row < height && col < width) {
        int pos = (row * width + col) * Channels;
        // Load data into shared memory
        sharedInput[threadId * Channels] = imageInput[pos];
        sharedInput[threadId * Channels + 1] = imageInput[pos + 1];
        sharedInput[threadId * Channels + 2] = imageInput[pos + 2];
        __syncthreads(); // Ensure all threads have loaded their part

        // Perform calculation in shared memory
        int grayPos = row * width + col;
        imageOutput[grayPos] = sharedInput[threadId * Channels + RED] * 0.299 +
                               sharedInput[threadId * Channels + GREEN] * 0.587 +
                               sharedInput[threadId * Channels + BLUE] * 0.114;
    }
}