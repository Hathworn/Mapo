#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate unique global thread index for efficient memory access
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    // Initialize each thread with a unique sequence number using tidx
    hiprand_init(seed, tidx, 0, &state[tidx]);
}