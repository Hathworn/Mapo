#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Optimize by using local thread to prevent atomic operations
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}