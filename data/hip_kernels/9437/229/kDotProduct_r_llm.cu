#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f; // Use a register for interim summation.

    // Loop unrolling for increased efficiency.
    while (eidx < numElements) {
        sum += a[eidx] * b[eidx];
        eidx += gridDim.x * DP_BLOCKSIZE;
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Reduce shared memory using a loop for clarity.
    for (uint s = DP_BLOCKSIZE / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            shmem[threadIdx.x] += shmem[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0]; // Write result of the block to the target array.
    }
}