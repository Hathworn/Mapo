#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * blockDim.x + threadIdx.x;

    if (tx < numCases) {
        const int label = __float2int_rn(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        // Use fast math function for log
        labelLogProbs[tx] = __logf(labelp);

        // Initialize correctProbs to 0
        correctProbs[tx] = 0.0f;
        
        // Check if the label probability is equal to the maximum probability
        if (labelp == maxp) {
            int numMax = 0;
            // Loop unrolling for potential speed-up
            for (int i = 0; i < numOut; i += 2) {
                numMax += (probs[i * numCases + tx] == maxp);
                if (i + 1 < numOut) {
                    numMax += (probs[(i + 1) * numCases + tx] == maxp);
                }
            }
            // Calculate probability for correct classification
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}