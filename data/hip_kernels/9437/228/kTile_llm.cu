#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate a single thread index for 2D grid
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    // Iterate over the output using the single thread index
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Directly compute source indices using modulo operations
        const uint srcY = (i / tgtWidth) % srcHeight;
        const uint srcX = (i % tgtWidth) % srcWidth;
        tgt[i] = src[srcY * srcWidth + srcX]; // Efficiently map tgt to src
    }
}