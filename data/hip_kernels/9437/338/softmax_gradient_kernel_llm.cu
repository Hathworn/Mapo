#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel( const int dim, const float* Y, const float* dY, float* dX) {
    // Offset pointers to the current block
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // Unroll the loop to improve performance
    for (int i = idx; i < dim; i += blockDim.x * 4) {
        if (i < dim) tmp += dY[i] * Y[i];
        if (i + blockDim.x < dim) tmp += dY[i + blockDim.x] * Y[i + blockDim.x];
        if (i + 2 * blockDim.x < dim) tmp += dY[i + 2 * blockDim.x] * Y[i + 2 * blockDim.x];
        if (i + 3 * blockDim.x < dim) tmp += dY[i + 3 * blockDim.x] * Y[i + 3 * blockDim.x];
    }
    
    // Store partial results in shared memory
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Perform parallel reduction to sum results
    if (idx < 128) reduction_buffer[idx] += reduction_buffer[idx + 128];
    __syncthreads();
    if (idx < 64) reduction_buffer[idx] += reduction_buffer[idx + 64];
    __syncthreads();
    if (idx < 32) {
        // Utilize warp-level intrinsic reductions
        tmp = reduction_buffer[idx] + reduction_buffer[idx + 32];
        tmp += reduction_buffer[idx + 16];
        tmp += reduction_buffer[idx + 8];
        tmp += reduction_buffer[idx + 4];
        tmp += reduction_buffer[idx + 2];
        tmp += reduction_buffer[idx + 1];
        reduction_buffer[idx] = tmp;
    }
    __syncthreads();
    
    // Compute gradient using the reduced value
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x * 4) {
        if (i < dim) dX[i] = Y[i] * (dY[i] - tmp);
        if (i + blockDim.x < dim) dX[i + blockDim.x] = Y[i + blockDim.x] * (dY[i + blockDim.x] - tmp);
        if (i + 2 * blockDim.x < dim) dX[i + 2 * blockDim.x] = Y[i + 2 * blockDim.x] * (dY[i + 2 * blockDim.x] - tmp);
        if (i + 3 * blockDim.x < dim) dX[i + 3 * blockDim.x] = Y[i + 3 * blockDim.x] * (dY[i + 3 * blockDim.x] - tmp);
    }
}