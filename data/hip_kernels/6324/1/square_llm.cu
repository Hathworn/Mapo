#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(float * d_out, float * d_in) {
    // Use blockIdx and blockDim for accessing a global memory index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within bounds for safety
    if (idx < N) {
        float f = d_in[idx];
        d_out[idx] = f * f;
    }
}