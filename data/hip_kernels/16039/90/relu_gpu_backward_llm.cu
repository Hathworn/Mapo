#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    // Calculate thread id
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use memory coalescing and boundary check
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        ingrad[i] = indata[i] > 0 ? outgrad[i] : 0;
    }
}