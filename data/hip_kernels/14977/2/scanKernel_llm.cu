#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Number of elements to put in the test array
#define TEST_SIZE 16
#define NUM_BINS 10

////////////////////////////////////////////////////////////////
////////////////// COPY EVERYTHING BELOW HERE //////////////////
////////////////////////////////////////////////////////////////

// Number of threads per block (1-d blocks)
#define BLOCK_WIDTH 4
// Functions to reduce with
#define ADD 0
#define MIN 1
#define MAX 2
// Device functions

__global__ void scanKernel(unsigned int* d_cdf, unsigned int* d_input, const size_t array_size)
{
    __shared__ unsigned int temp[BLOCK_WIDTH << 1];
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int index = BLOCK_WIDTH * bx + tx;

    // Load data into shared memory
    if (2 * index < array_size) {
        temp[2 * tx] = d_input[2 * index];
        temp[2 * tx + 1] = (2 * index + 1 < array_size) ? d_input[2 * index + 1] : 0;
    }
    __syncthreads();

    int offset = 1;

    // Up-sweep/reduction step
    for (int powOf2 = BLOCK_WIDTH; powOf2 > 0; powOf2 >>= 1) {
        if (tx < powOf2) {
            int idx1 = offset * (2 * tx + 1) - 1;
            int idx2 = offset * (2 * tx + 2) - 1;
            temp[idx2] += temp[idx1];
        }
        offset <<= 1;
        __syncthreads();
    }

    if (tx == 0) {
        temp[2 * BLOCK_WIDTH - 1] = 0; // Reset the last element
    }
    __syncthreads();

    offset >>= 1;

    // Down-sweep step
    for (int powOf2 = 1; powOf2 < BLOCK_WIDTH * 2; powOf2 <<= 1) {
        if (tx < powOf2) {
            int idx1 = offset * (2 * tx + 1) - 1;
            int idx2 = offset * (2 * tx + 2) - 1;
            unsigned int t = temp[idx1];
            temp[idx1] = temp[idx2];
            temp[idx2] += t;
        }
        offset >>= 1;
        __syncthreads();
    }

    // Write results back to global memory
    if (2 * index < array_size) {
        d_cdf[2 * index] = temp[2 * tx];
        if (2 * index + 1 < array_size) {
            d_cdf[2 * index + 1] = temp[2 * tx + 1];
        }
    }
}