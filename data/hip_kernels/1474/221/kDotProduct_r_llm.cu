#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];
    
    uint tid = threadIdx.x;
    uint eidx = DP_BLOCKSIZE * blockIdx.x + tid;
    shmem[tid] = 0.0f;
    
    // Unroll the loop to improve memory throughput
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        shmem[tid] += a[eidx] * b[eidx];
    }

    __syncthreads();

    // Use a loop for the reductions to improve readability and conciseness
    for (uint stride = DP_BLOCKSIZE / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shmem[tid] += shmem[tid + stride];
        }
        __syncthreads();
    }
    
    // The final result is stored only by the first thread in the block
    if (tid == 0) {
        target[blockIdx.x] = shmem[0];
    }
}