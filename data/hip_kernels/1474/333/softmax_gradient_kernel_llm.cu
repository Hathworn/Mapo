#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    extern __shared__ float reduction_buffer[];  // Use dynamic shared memory

    // Compute global linear index
    int global_idx = blockIdx.x * dim + threadIdx.x;

    float sum = 0.0f;
    // Unroll loop to reduce iterations and improve performance
    for (int i = threadIdx.x; i < dim; i += blockDim.x) {
        sum += dY[global_idx + i] * Y[global_idx + i];
    }
    
    reduction_buffer[threadIdx.x] = sum;
    __syncthreads();

    // Efficient parallel reduction within the block
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            reduction_buffer[threadIdx.x] += reduction_buffer[threadIdx.x + stride];
        }
        __syncthreads();
    }

    float total_sum = reduction_buffer[0];
    
    // Calculate gradient using shared total_sum
    for (int i = threadIdx.x; i < dim; i += blockDim.x) {
        dX[global_idx + i] = Y[global_idx + i] * (dY[global_idx + i] - total_sum);
    }
}