#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread index 0 to perform the division to avoid race conditions.
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}