#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Compute global thread index more efficiently
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    // Initialize CURAND state with minimal function call
    hiprand_init(seed, tidx, 0, &state[tidx]);
}