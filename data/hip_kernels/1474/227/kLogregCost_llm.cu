#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        // Use fast math log function
        labelLogProbs[tx] = __logf(labelp);

        // Simplify conditional assignment using ternary operator
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            float maxProb = maxProbs[tx];
            // Optimize loop by unrolling it or minimizing the iterator operations
            for (int i = 0; i < numOut; ++i) {
                numMax += (probs[i * numCases + tx] == maxProb);
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}