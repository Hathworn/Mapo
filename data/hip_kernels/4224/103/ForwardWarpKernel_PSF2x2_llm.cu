#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (i >= h || j >= w) return;

    // Precompute commonly used indices
    int flow_row_offset  = i * flow_stride + j;
    int image_row_offset = i * image_stride + j;

    // Compute warped position
    float cx = u[flow_row_offset] * time_scale + j + 1.0f;
    float cy = v[flow_row_offset] * time_scale + i + 1.0f;

    // Decompose into integer and fractional parts
    float px, py, dx = modff(cx, &px), dy = modff(cy, &py);

    // Integer target pixel coords
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);

    // Fetch source value once
    float value = src[image_row_offset];

    // Consolidate the four possible updates into a loop to enhance readability and maintainability
    #pragma unroll
    for (int corner = 0; corner < 4; ++corner) 
    {
        int offset_x = (corner & 1) ? 0 : -1;
        int offset_y = (corner & 2) ? 0 : -1;
        float weight_x = (corner & 1) ? dx : (1.0f - dx);
        float weight_y = (corner & 2) ? dy : (1.0f - dy);
        int final_tx = tx + offset_x;
        int final_ty = ty + offset_y;
        
        if (!(final_tx >= w || final_tx < 0 || final_ty >= h || final_ty < 0)) {
            float weight = weight_x * weight_y;
            atomicAdd(dst + final_ty * image_stride + final_tx, value * weight);
            atomicAdd(normalization_factor + final_ty * image_stride + final_tx, weight);
        }
    }
}