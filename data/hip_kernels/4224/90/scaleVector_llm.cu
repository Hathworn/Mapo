#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Use fast math operation for better performance
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos < len) {
        d_res[pos] = __fmul_rn(d_src[pos], scale); // fmul_rn for faster multiplication
    }
}