#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Use shared memory to reduce global memory access
    __shared__ float shared_scale[BLOCK_DIM_X][BLOCK_DIM_Y];

    if (i < h && j < w) {  // Combine conditions for efficiency
        const int pos = i * s + j;

        // Load normalization factor into shared memory
        shared_scale[threadIdx.y][threadIdx.x] = normalization_factor[pos];
        __syncthreads();  // Ensure all threads have loaded data

        float scale = shared_scale[threadIdx.y][threadIdx.x];

        // Avoid branch, use conditional in calculation
        image[pos] *= (scale != 0.0f) ? (1.0f / scale) : 1.0f;
    }
}