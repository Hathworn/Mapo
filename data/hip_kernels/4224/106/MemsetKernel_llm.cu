#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate a unique index for each thread
    int index = blockIdx.y * blockDim.y * w + blockIdx.x * blockDim.x + threadIdx.y * w + threadIdx.x;

    // Flattened index is valid only within the bounds
    if (index < w * h) {
        image[index] = value;
    }
}