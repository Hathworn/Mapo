#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kEltwiseLogregCost(float* predmap, float* indmap, float* indlogpred, float* correctprobs, int numCases, int numTasks, int per_thread_case) {
    const int task_id = blockIdx.x;
    const int start_tx = threadIdx.x * per_thread_case;
    const int end_tx = min(start_tx + per_thread_case, numCases);
    const float EPSILON = 1e-20f; // Minimum value allowed, avoid log( 0 )

    // Return early if task_id is out of bounds
    if (task_id >= numTasks) {
        return;
    }

    for (int c_id = start_tx; c_id < end_tx; ++c_id) {
        int pos = task_id * numCases + c_id;
        float pred_value = predmap[pos];
        float t = __fdividef(1.0f, 1.0f + __expf(-pred_value)); // Calculate logistic sigmoid

        // Use branchless approach to handle 'indmap == 1' condition
        float is_indmap_one = indmap[pos] * 2.0f - 1.0f; // -1.0f if 0, 1.0f if 1
        float adjusted_t = (1.0f + is_indmap_one) * t - is_indmap_one; // Efficient conditional calculation

        // Ensure adjusted_t is not less than EPSILON
        adjusted_t = fmaxf(adjusted_t, EPSILON);

        // Calculate log probability and store
        indlogpred[pos] = __logf(adjusted_t);

        // Store the final correct probability
        correctprobs[pos] = adjusted_t;
    }
}