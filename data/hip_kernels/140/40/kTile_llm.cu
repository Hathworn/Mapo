#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate global index for each thread
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    // Use stride loop for efficient memory access
    for (uint i = idx; i < tgtWidth * tgtHeight; i += stride) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Directly mapping source to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}