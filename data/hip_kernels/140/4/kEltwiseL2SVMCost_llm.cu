#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kEltwiseL2SVMCost(float* ydata, float* ldata, float* pre_grad, float* all_cost, float a, float b, int numCases, int numTasks, int per_thread_case) {
    const int task_id = blockIdx.x;
    const int start_tx = threadIdx.x * per_thread_case;
    const int end_tx = min(start_tx + per_thread_case, numCases);
    
    if (task_id >= numTasks) {
        return;
    }
    
    // Precompute the base index to reduce redundant calculations
    int base_pos = task_id * numCases;

    for (int c_id = start_tx; c_id < end_tx; ++c_id) {
        int pos = base_pos + c_id;
        float yldata_diff = ldata[pos] - b; // Share load into memory
        float tmp = fmaxf(a - ydata[pos] * yldata_diff, 0);
        
        pre_grad[pos] = tmp;
        all_cost[pos] = tmp * tmp;
    }
}