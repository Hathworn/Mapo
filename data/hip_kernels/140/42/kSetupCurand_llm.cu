#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate global thread index
    const uint tidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use HIP built-in intrinsics for efficiency
    hiprand_init(seed, tidx, 0, &state[tidx]);
}