#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZE 1024

__global__ void vectorAdd(int *a, int *b, int *c, int n)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    // Ensure thread is within bounds
    if(i < n) 
        c[i] = a[i] + b[i];
}