#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_vecDouble(int *in, int *out, const int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global index
    if (i < n) {
        out[i] = in[i] * 2;  // Double the element
    }
}