#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 256

__global__ void matrix_mult(float *dev_a, float* dev_b, float* dev_c, int Width) {
    // Use shared memory for tiling
    __shared__ float tile_a[16][16];
    __shared__ float tile_b[16][16];

    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    float Pvalue = 0;

    // Loop over the tiles of the input matrices
    for (int m = 0; m < Width/blockDim.x; ++m) {
        // Collaborative loading of tiles into shared memory
        tile_a[threadIdx.y][threadIdx.x] = dev_a[Row * Width + m * blockDim.x + threadIdx.x];
        tile_b[threadIdx.y][threadIdx.x] = dev_b[(m * blockDim.x + threadIdx.y) * Width + Col];

        // Ensure all threads have loaded the tiles
        __syncthreads();

        // Multiply the tiles together
        for (int k = 0; k < blockDim.x; ++k) {
            Pvalue += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }

        // Synchronize to make sure that the preceding computations are done before loading new tiles
        __syncthreads();
    }

    if ((Row < Width) && (Col < Width)) {
        dev_c[Row * Width + Col] = Pvalue;
    }
}