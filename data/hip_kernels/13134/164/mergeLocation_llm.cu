#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure valid index and process in bounds
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];
        float scaled_x = loc.x * scale;
        float scaled_y = loc.y * scale;
        
        // Coalesced memory writes
        x[ptidx] = scaled_x;
        y[ptidx] = scaled_y;
    }
}