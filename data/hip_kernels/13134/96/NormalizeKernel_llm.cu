#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate the 1D index rather than 2D index to improve memory access pattern
    int pos = (blockIdx.y * blockDim.y + threadIdx.y) * s + (blockIdx.x * blockDim.x + threadIdx.x);
    
    // Use a logical check to avoid accessing out-of-bounds memory
    if (pos < w * h) {
        float scale = normalization_factor[pos];
        
        // Replace conditional operator for branch-less inversion calculation 
        float invScale = rsqrt(scale * scale + 1e-8); 
        
        image[pos] *= invScale;
    }
}