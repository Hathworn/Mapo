#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    // Casting char pointers to unsigned int pointers for aligned writes
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int* orig_ptr = ptr;

    unsigned int pattern = 1;
    unsigned long mask = 4;

    *ptr = pattern;

    // Improved loop iteration to reduce unnecessary calculations
    while (ptr < end_ptr) {
        // Compute target position with mask using bitwise OR
        unsigned int* next_ptr = (unsigned int*)(((unsigned long)orig_ptr) | mask);

        if (next_ptr >= end_ptr) {
            break;
        }

        // Write pattern to memory
        *next_ptr = pattern;

        // Update mask and pattern for next iteration
        // This increment step avoids repetitive computation
        mask <<= 1;
        pattern <<= 1;
        orig_ptr = next_ptr;
    }
}