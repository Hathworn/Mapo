#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    // Calculate thread-specific memory address calculation
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE + threadIdx.x * sizeof(unsigned int));

    // Ensure thread-specific pointer does not exceed end_ptr
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Write pattern in a loop across threadIdx within BLOCKSIZE
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        ptr[i] = pattern;
    }
}