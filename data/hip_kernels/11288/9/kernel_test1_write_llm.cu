#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    unsigned long* ptr = (unsigned long*)(_ptr + blockIdx.x * BLOCKSIZE);

    // Early exit if the pointer exceeds the end pointer
    if (ptr >= (unsigned long*)end_ptr) {
        return;
    }

    // Use a thread-wise stride to increase memory access coalescing
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned long); i += blockDim.x) {
        ptr[i] = (unsigned long)&ptr[i];
    }
}