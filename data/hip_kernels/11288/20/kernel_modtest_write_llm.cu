#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    // Calculate the pointer once at the start
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Avoid unnecessary computation when out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Precompute limit to avoid repeated division
    unsigned int limit = BLOCKSIZE / sizeof(unsigned int);

    // Unrolling loop to optimize memory writes
    #pragma unroll
    for (unsigned int i = offset; i < limit; i += MOD_SZ) {
        ptr[i] = p1;
    }

    // Optimize second loop by reducing conditional checks
    #pragma unroll
    for (unsigned int i = 0; i < limit; i++) {
        if (i % MOD_SZ != offset) {
            ptr[i] = p2;
        }
    }
}