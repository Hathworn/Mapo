#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    // Calculate initial pointers based on block index
    unsigned int* orig_ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    if (orig_ptr >= (unsigned int*) end_ptr) {
        return;
    }
    
    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);
    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Write initial pattern to original pointer address
    *orig_ptr = pattern;

    // Loop with optimized pointer arithmetic
    while (true) {
        unsigned int* ptr = (unsigned int*)((unsigned long)orig_ptr | mask);
        if (ptr == orig_ptr) {
            mask <<= 1; // Increment mask
            continue;
        }
        if (ptr >= block_end) {
            break; // Exit loop if out of bounds
        }

        *ptr = pattern; // Write pattern at calculated pointer
        pattern <<= 1;  // Increment pattern
        mask <<= 1;     // Increment mask
    }
}