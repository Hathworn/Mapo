#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Use shared memory to cache pattern updates
    __shared__ unsigned int shared_pattern;

    unsigned int k = offset;
    shared_pattern = pattern;  // Initialize shared pattern

    unsigned int i = threadIdx.x;
    unsigned int stride = blockDim.x;

    // Loop with stride of block size for possible unroll by compiler
    for (; i < BLOCKSIZE / sizeof(unsigned int); i += stride) {
        ptr[i] = shared_pattern;  // Write cached pattern
        k++;
        if (k >= 32) {
            k = 0;
            shared_pattern = lb;
        } else {
            shared_pattern = (shared_pattern << 1) | sval;  // Update pattern
        }
    }
}