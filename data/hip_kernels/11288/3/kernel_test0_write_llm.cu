#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Early exit if the pointer is out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Utilize threadIdx.x for parallel writing within BLOCKSIZE
    int idx = threadIdx.x;
    int stride = blockDim.x;

    for (int i = idx; i < BLOCKSIZE / sizeof(unsigned int); i += stride){
        ptr[i] = pattern;
    }

    return;
}