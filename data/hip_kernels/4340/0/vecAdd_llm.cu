#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    // Efficient vector addition by ensuring coalesced memory access
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < len) {
        out[i] = in1[i] + in2[i];
    }
}