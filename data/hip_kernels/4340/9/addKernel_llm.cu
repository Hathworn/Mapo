#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global index using block and thread indices
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform addition only if index is within bounds
    if (i < N)  // Assume N is the size of the arrays
    {
        c[i] = a[i] + b[i];
    }
}