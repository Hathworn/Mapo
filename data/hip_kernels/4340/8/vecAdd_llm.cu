#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(float * in1, int offset, int len) {
//@@ Optimized vector addition using loop unrolling
int i = threadIdx.x + blockIdx.x * blockDim.x;

if (offset + i < len) {
    in1[offset + i] += in1[offset - 1];
}
if (offset + i + blockDim.x < len) {
    in1[offset + i + blockDim.x] += in1[offset - 1];
}
}