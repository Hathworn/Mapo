#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan(float * input, float * output,  int len) {
    // Load a segment of the input vector into shared memory
    __shared__ float sh_input[2048];

    int tx = threadIdx.x;
    int bdimx = blockDim.x;
    int i = 2 * blockIdx.x * blockDim.x + tx;

    int start = 2 * blockIdx.x * blockDim.x;
    int Col1 = start + tx;
    int Col2 = start + bdimx + tx;

    // Coalesced global memory access and reduce control divergence
    sh_input[tx] = (Col1 < len) ? input[Col1] : 0.0f;
    sh_input[tx + bdimx] = (Col2 < len) ? input[Col2] : 0.0f;

    __syncthreads();

    unsigned int stride; 
    int index;
    // Traverse the reduction tree down
    for (stride = 1; stride <= 2 * bdimx; stride *= 2) {
        index = (tx + 1) * stride * 2 - 1;
        if (index < 2 * bdimx) 
            sh_input[index] += sh_input[index - stride];
        __syncthreads();
    }

    // Traverse the reduction tree up
    for (stride = bdimx / 2; stride > 0; stride /= 2) {
        __syncthreads();
        index = (tx + 1) * stride * 2 - 1;
        if (index + stride < 2 * bdimx) 
            sh_input[index + stride] += sh_input[index];
    }

    // Write the computed sum of the block to the output vector at the correct index
    __syncthreads();
    if (i < len) // Bounds checking to avoid out-of-bounds writes
        output[i] = sh_input[tx];
    if (i + bdimx < len)
        output[i + bdimx] = sh_input[tx + bdimx];
}