#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMultiply(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    // Calculate thread row and column within the C matrix
    int iRow = blockIdx.y * blockDim.y + threadIdx.y;
    int iCol = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread computes only valid entries
    if (iRow < numCRows && iCol < numCColumns) {
        float Cvalue = 0.0;
        
        // Use shared memory for tiles
        __shared__ float As[16][16];
        __shared__ float Bs[16][16];
        
        for (int k = 0; k < (numAColumns + 15) / 16; ++k) {
            // Load tiles into shared memory
            if (iRow < numARows && k * 16 + threadIdx.x < numAColumns) 
                As[threadIdx.y][threadIdx.x] = A[iRow * numAColumns + k * 16 + threadIdx.x];
            else
                As[threadIdx.y][threadIdx.x] = 0.0;
            
            if (k * 16 + threadIdx.y < numBRows && iCol < numBColumns) 
                Bs[threadIdx.y][threadIdx.x] = B[(k * 16 + threadIdx.y) * numBColumns + iCol];
            else
                Bs[threadIdx.y][threadIdx.x] = 0.0;
            
            __syncthreads();
            
            // Compute partial product
            for (int n = 0; n < 16; ++n) {
                Cvalue += As[threadIdx.y][n] * Bs[n][threadIdx.x];
            }
            
            __syncthreads();
        }
        
        // Write result to C matrix
        C[iRow * numCColumns + iCol] = Cvalue;
    }
}