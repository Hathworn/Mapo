#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    // Use shared memory for data locality and reduced global memory access
    extern __shared__ float shared_data[]; 
    float *shared_in1 = shared_data;
    float *shared_in2 = shared_data + blockDim.x;
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < len) {
        // Load data into shared memory
        shared_in1[threadIdx.x] = in1[i];
        shared_in2[threadIdx.x] = in2[i];
        __syncthreads();
        
        // Perform the addition using shared memory
        out[i] = shared_in1[threadIdx.x] + shared_in2[threadIdx.x];
    }
}