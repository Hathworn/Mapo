#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16  // Define TILE_WIDTH for clarity

__global__ void matrixMultiply(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    // Optimizing shared memory allocation
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float Cvalue = 0;

    // Loop over the A and B tiles required to compute the C element
    for (int t = 0; t < (numAColumns - 1) / TILE_WIDTH + 1; ++t) {
        // Avoid unnecessary divergence by loading within bounds
        if (Row < numARows && t * TILE_WIDTH + tx < numAColumns) {
            ds_A[ty][tx] = A[Row * numAColumns + t * TILE_WIDTH + tx];
        } else {
            ds_A[ty][tx] = 0.0;
        }

        if (t * TILE_WIDTH + ty < numBRows && Col < numBColumns) {
            ds_B[ty][tx] = B[(t * TILE_WIDTH + ty) * numBColumns + Col];
        } else {
            ds_B[ty][tx] = 0.0;
        }

        __syncthreads();

        // Use loop unrolling for better performance on matrix multiplication
        #pragma unroll
        for (int i = 0; i < TILE_WIDTH; ++i) {
            Cvalue += ds_A[ty][i] * ds_B[i][tx];
        }
        __syncthreads();
    }
  
    if (Row < numARows && Col < numBColumns) {
        C[Row * numBColumns + Col] = Cvalue;
    }
}