#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global index based on block and thread indices
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index does not exceed allocated memory
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}