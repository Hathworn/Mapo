#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __l1dist(double *A, int lda, double *B, int ldb, double *C, int ldc, int d, int nrows, int ncols, double p) {
    // Calculate global row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure row and column are within matrix bounds
    if (row < nrows && col < ncols) {
        double sum = 0.0;

        // Compute L1 distance for the current element
        for (int k = 0; k < d; ++k) {
            sum += fabs(A[row * lda + k] - B[k * ldb + col]);
        }

        // Store the computed distance into the matrix C
        C[row * ldc + col] = sum;
    }
}