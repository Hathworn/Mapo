#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __radixcounts(double *a, int n, int digit, unsigned int *bi) {
    __shared__ unsigned int ic[RNDVALS];

    // Calculate the range of data each block is responsible for
    int istart = ((long long)blockIdx.x * n) / gridDim.x;
    int iend = ((long long)(blockIdx.x + 1) * n) / gridDim.x;
    int tid = threadIdx.x;
    int bibase = RNDVALS * (blockIdx.x + istart / RBIGBLK);

    for (int i = istart; i < iend; i += RBIGBLK) {
        __syncthreads();
        ic[tid] = 0;  // Initialize shared memory
        __syncthreads();

        // Efficiently loop through elements with strided access pattern
        for (int j = i + tid; j < min(iend, i + RBIGBLK); j += blockDim.x) {
            double v = a[j];
            unsigned char *cv = (unsigned char *)&v;
            atomicInc(&ic[cv[digit]], 65536 * 32767);
        }
        __syncthreads();

        // Store results back to global memory
        bi[bibase + tid] = ic[tid];
        bibase += RNDVALS;
    }
}