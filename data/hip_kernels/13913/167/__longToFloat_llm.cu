#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __longToFloat(long long *A, float *B, int N) {

    // Calculate the global index for this thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 
              blockIdx.y * gridDim.x * blockDim.x; 

    if (idx < N) { // Ensure the index is within bounds
        B[idx] = static_cast<float>(A[idx]); // Convert long to float
    }
}