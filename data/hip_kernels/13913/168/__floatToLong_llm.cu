#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __floatToLong(float *A, long long *B, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride pattern for coalesced memory access and efficient iteration
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < N; i += stride) {
        B[i] = static_cast<long long>(A[i]);  // Explicit cast for clarity
    }
}