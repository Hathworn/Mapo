#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __minkowskidist(float *A, int lda, float *B, int ldb, float *C, int ldc, int d, int nrows, int ncols, float p) {
    // Calculate thread indices for matrix rows and columns
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check boundaries
    if (row < nrows && col < ncols) {
        float sum = 0.0f;
        // Compute the Minkowski distance for each dimension
        for (int i = 0; i < d; i++) {
            float diff = A[row * lda + i] - B[i * ldb + col];
            sum += pow(fabs(diff), p);
        }
        // Compute p-th root of sum for the Minkowski distance
        C[row * ldc + col] = pow(sum, 1.0f / p);
    }
}