#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __full(int *ir, int *ic, float *data, float *od, int nrows, int ncols, int nnz) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    // Use id and stride to allow multiple iterations by the same thread
    for (int i = id; i < nnz; i += stride) {
        int row = ir[i]; // Cache ir[] in register
        int col = ic[i]; // Cache ic[] in register
        float v = data[i]; // Cache data[] in register
        od[row + col * nrows] = v; // Directly assign to output
    }
}