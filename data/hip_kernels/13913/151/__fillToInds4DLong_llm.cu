#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __fillToInds4DLong(long long A, long long *B, int ldb, int rdb, int tdb, int *I, int nrows, int *J, int ncols, int *K, int nk, int *L, int nl) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = blockDim.x * gridDim.x;
    int idx, mapi, mapj, mapk, mapl;
    
    for (idx = tid; idx < nrows * ncols * nk * nl; idx += gridStride) {
        // Calculate 4D index from flat index
        int l = idx / (nrows * ncols * nk);
        int rem = idx % (nrows * ncols * nk);
        int k = rem / (nrows * ncols);
        rem = rem % (nrows * ncols);
        int j = rem / nrows;
        int i = rem % nrows;
        
        // Apply mapping if necessary
        mapl = (L != NULL) ? L[l] : l;
        mapk = (K != NULL) ? K[k] : k;
        mapj = (J != NULL) ? J[j] : j;
        mapi = (I != NULL) ? I[i] : i;
        
        // Store the value
        B[mapi + ldb * (mapj + rdb * (mapk + tdb * mapl))] = A;
    }
}