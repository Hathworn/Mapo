#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __fillToInds3D(float A, float *B, int ldb, int rdb, int *I, int nrows, int *J, int ncols, int *K, int nk) {
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x; 
    int step = blockDim.x * gridDim.x;
    
    // Loop to update B at specified indices
    for (int id = tid; id < nrows * ncols * nk; id += step) {
        int k = id / (nrows * ncols);
        int remainder = id % (nrows * ncols);
        int j = remainder / nrows;
        int i = remainder % nrows;
        
        // Map indices through I, J, K arrays if they exist
        int mapi = (I != NULL) ? I[i] : i;
        int mapj = (J != NULL) ? J[j] : j;
        int mapk = (K != NULL) ? K[k] : k;
        
        // Efficient memory access to update B
        B[mapi + ldb * (mapj + rdb * mapk)] = A;
    }
}