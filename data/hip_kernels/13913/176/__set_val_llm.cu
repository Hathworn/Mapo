#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __set_val(float *A, float val, int length) {
    int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
    int stride = blockDim.x * gridDim.x * gridDim.y; // Calculate stride outside loop
    for (int i = ip; i < length; i += stride) { // Use pre-calculated stride
        A[i] = val;
    }
}