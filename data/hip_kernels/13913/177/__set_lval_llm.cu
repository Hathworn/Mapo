#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __set_lval(long long *A, long long val, int length) {
    // Calculate global thread index
    int ip = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Use stride for better occupancy and avoid warp divergence
    int stride = blockDim.x * gridDim.x * gridDim.y;

    // Loop with updated stride logic
    for (int i = ip; i < length; i += stride) {
        A[i] = val;
    }
}