#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __initSeq2(int *A, int nrows, int ncols) {
    int idx = blockIdx.x + gridDim.x * blockIdx.y; // 2D block index
    int tid = threadIdx.x; // Thread index within a block
    int blockSize = blockDim.x * gridDim.x * gridDim.y; // Total threads
    int i = tid + blockDim.x * idx; // Linear index

    // Unroll loop for better performance
    int stride = blockSize;
    while (i < nrows * ncols) {
        A[i] = i / nrows;
        i += stride;
    }
}