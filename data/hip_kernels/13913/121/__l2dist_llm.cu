#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __l2dist(float *A, int lda, float *B, int ldb, float *C, int ldc, int d, int nrows, int ncols, float p) {
    int row = blockIdx.x * blockDim.x + threadIdx.x; // Calculate row index
    int col = blockIdx.y * blockDim.y + threadIdx.y; // Calculate column index

    if (row < nrows && col < ncols) {
        float dist = 0.0f; // Initialize distance
        for (int k = 0; k < d; ++k) {
            float diff = A[row * lda + k] - B[k * ldb + col]; // Element-wise subtraction
            dist += diff * diff; // Accumulate squared differences
        }
        C[row * ldc + col] = powf(dist, 1.0f / p); // Assign computed distance
    }
}