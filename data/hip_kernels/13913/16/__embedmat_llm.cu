#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __embedmat(float *a, int *b, long long *c, int n) {
    // Calculate grid stride to allow entire grid to participate in computation
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; 
    const int signbit = 0x80000000;
    const int mag = 0x7fffffff;

    // Use grid-stride loop for accessing elements
    for (int i = tid; i < n; i += stride) {
        float v = a[i];
        int vi = *((int*)&v);
        if (vi & signbit) {
            vi = -(vi & mag);
        }
        c[i] = (long long)vi + (((long long)b[i]) << 32);
    }
}