#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __toInt(double *A, int *B, int N) {
    // Calculate the global thread index for 2D grid
    int ip = blockIdx.x + gridDim.x * blockIdx.y;
    int idx = threadIdx.x + blockDim.x * ip; // Global thread index
    int stride = blockDim.x * gridDim.x * gridDim.y; // Stride for iterating over elements

    for (int i = idx; i < N; i += stride) {
        B[i] = static_cast<int>(A[i]);
    }
}