#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __stratifycounts(float *strata, int n,  float *a, unsigned int *bi) {
    __shared__ unsigned int ic[SNDVALS][SNDGRPS];
    __shared__ float ss[SNDVALS];
    int istart = (int)(((long long)blockIdx.x) * n / gridDim.x);
    int iend = (int)(((long long)(blockIdx.x+1)) * n / gridDim.x);
    int bibase = SNDVALS * (blockIdx.x + istart / SBIGBLK);
    int tid = threadIdx.x + threadIdx.y * blockDim.x;

    // Load strata to shared memory by the first warp
    if (threadIdx.y == 0 && threadIdx.x < SNDVALS) {
        ss[threadIdx.x] = strata[threadIdx.x];
    }
    __syncthreads();

    for (int i = istart; i < iend; i += SBIGBLK) {
        // Initialize shared memory histogram bins
        if (threadIdx.y < SNDGRPS && threadIdx.x < SNDVALS) {
            ic[threadIdx.x][threadIdx.y] = 0;
        }
        __syncthreads();
        
        for (int k = i + tid; k < min(iend, i + SBIGBLK); k += SNTHREADS) {
            float v = a[k];
            int j = 0;
            
            // Unroll the loop for stratum calculation
            #pragma unroll
            for (int idx = 0; idx < 8; ++idx) {
                j = (v > ss[j]) ? 2*j+2 : 2*j+1;
            }
            
            j = j - SNDVALS + 1;
            atomicAdd(&ic[j][threadIdx.y], 1);  // Use atomicAdd instead of atomicInc
        }
        __syncthreads();
        
        // Sum histogram bins using first column of threads
        if (threadIdx.y == 0 && threadIdx.x < SNDVALS) {
            unsigned int sum = 0;
            #pragma unroll
            for (int idx = 0; idx < SNDGRPS; ++idx) {
                sum += ic[threadIdx.x][idx];
            }
            bi[bibase + threadIdx.x] = sum;
        }
        bibase += SNDVALS;
    }
}