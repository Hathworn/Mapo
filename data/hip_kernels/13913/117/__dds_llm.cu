#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __dds(int nrows, int nnz, float *A, float *B, int *Cir, int *Cic, float *P) {
    // Use a dynamic shared memory size
    extern __shared__ float parts[];

    // Calculate the range of work for each block
    int jstart = ((long long)blockIdx.x) * nnz / gridDim.x;
    int jend = ((long long)(blockIdx.x + 1)) * nnz / gridDim.x;

    // Global thread ID
    int tid = threadIdx.x + blockDim.x * threadIdx.y;

    // Use coalesced memory access pattern
    for (int j = jstart; j < jend ; j++) {
        float sum = 0;
        int aoff = nrows * Cir[j];
        int boff = nrows * Cic[j];

        // Optimize memory access by ensuring aligned reads
        for (int i = tid; i < nrows; i += blockDim.x * blockDim.y) {
            sum += A[i + aoff] * B[i + boff];
        }

        // Store the partial sum in shared memory
        parts[tid] = sum;

        // Perform reduction using shared memory
        for (int offset = blockDim.x * blockDim.y / 2; offset > 0; offset /= 2) {
            __syncthreads();
            if (tid < offset) {
                parts[tid] += parts[tid + offset];
            }
        }

        __syncthreads();

        // Write the result of the reduction to global memory
        if (tid == 0) {
            P[j] = parts[0];
        }
        __syncthreads();
    }
}