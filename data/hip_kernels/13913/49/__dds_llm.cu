#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __dds(int nrows, int nnz, double *A, double *B, int *Cir, int *Cic, double *P) {
    extern __shared__ double parts[];  // Use dynamic shared memory
    int tid = threadIdx.x + blockDim.x * threadIdx.y;
    int jstart = ((long long)blockIdx.x) * nnz / gridDim.x;
    int jend = ((long long)(blockIdx.x + 1)) * nnz / gridDim.x;

    // Unroll j loop for better performance
    for (int j = jstart; j < jend; j += blockDim.x * blockDim.y) {
        double sum = 0;
        int aoff = nrows * Cir[j];
        int boff = nrows * Cic[j];

        // Use local variable for shared memory indexing
        int local_tid = tid + j % blockDim.x;
        
        for (int i = tid; i < nrows; i += blockDim.x * blockDim.y) {
            sum += A[i + aoff] * B[i + boff];
        }
        parts[local_tid] = sum;
        
        // Optimized reduction using loop unrolling
        for (int stride = blockDim.x * blockDim.y / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            if (local_tid < stride) {
                parts[local_tid] += parts[local_tid + stride];
            }
        }
        
        if (local_tid == 0) {
            P[j] = parts[0];
        }
    }
}