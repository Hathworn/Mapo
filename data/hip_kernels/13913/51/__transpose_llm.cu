#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __transpose(double *in, int instride, double *out, int outstride, int nrows, int ncols) {
    int nx = BLOCKDIM * gridDim.x;
    int ny = BLOCKDIM * gridDim.y;
    int ix = BLOCKDIM * blockIdx.x;
    int iy = BLOCKDIM * blockIdx.y;
    __shared__ double tile[BLOCKDIM][BLOCKDIM+1]; // Avoid bank conflicts with padding

    for (int yb = iy; yb < ncols; yb += ny) {
        for (int xb = ix; xb < nrows; xb += nx) {
            if (xb + threadIdx.x < nrows) {
                int ylim = min(ncols, yb + BLOCKDIM);
                // Coalesced read from global memory
                for (int y = threadIdx.y + yb; y < ylim; y += blockDim.y) {
                    tile[threadIdx.x][y-yb] = in[threadIdx.x+xb + y*instride];
                }
            }
            __syncthreads();

            if (yb + threadIdx.x < ncols) {
                int xlim = min(nrows, xb + BLOCKDIM);
                // Coalesced write to global memory
                for (int x = threadIdx.y + xb; x < xlim; x += blockDim.y) {
                    out[threadIdx.x + yb + x*outstride] = tile[x-xb][threadIdx.x];
                }
            }
            __syncthreads();
        }
    }
}