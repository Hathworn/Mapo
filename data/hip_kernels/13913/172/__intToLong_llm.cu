#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __intToLong(int *A, long long *B, int N) {
    int ip = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
    int stride = blockDim.x * gridDim.x * gridDim.y;
    
    // Unrolling loop for better performance
    for (int i = ip; i < N; i += stride) {
        B[i] = static_cast<long long>(A[i]);
    }
}