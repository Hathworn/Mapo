#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __extractmat(float *a, int *b, long long *c, int n) {
    int tid = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
    const int signbit = 0x80000000;
    const int mag = 0x7fffffff;
    int stride = blockDim.x * gridDim.x * gridDim.y;  // Calculate stride for better memory access

    for (int i = tid; i < n; i += stride) {
        int vi = __ldg((int *)&c[i]);  // Use __ldg for read-only access to improve performance
        if (vi & signbit) {
            vi = -(vi & mag);
        }
        a[i] = __int_as_float(vi);  // Use __int_as_float for type conversion
        b[i] = __ldg(((int *)&c[i]) + 1);  // Use __ldg for read-only access
    }
}