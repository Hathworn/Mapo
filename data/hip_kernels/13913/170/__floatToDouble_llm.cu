#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __floatToDouble(float *A, double *B, int N) {
    // Calculate the global thread index
    int ip = threadIdx.x + blockDim.x * blockIdx.x + blockDim.x * gridDim.x * blockIdx.y;

    // Use a stride loop for better memory access pattern
    int stride = blockDim.x * gridDim.x * gridDim.y;
    for (int i = ip; i < N; i += stride) {
        B[i] = static_cast<double>(A[i]);
    }
}