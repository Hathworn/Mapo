#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __fillToInds(double A, double *B, int *I, long long len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Simplify thread index calculation
    int stride = blockDim.x * gridDim.x; // Establish stride for looping

    for (long long i = tid; i < len; i += stride) { // Use simplified stride in loop
        B[I[i]] = A;
    }
}