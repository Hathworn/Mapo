#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __embedmat2d(float *a, long long *b, int nrows, int ncols, int sortdown) {
    // Calculate the global thread ID
    int tid = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);

    // Constants for handling sign bits
    const int signbit = 0x80000000;
    const int mag = 0x7fffffff;
    
    // Loop through each element this thread is responsible for
    for (int i = tid; i < nrows * ncols; i += blockDim.x * gridDim.x * gridDim.y) {
        // Read float value and interpret as integer
        float v = a[i];
        int vi = *((int *)&v);

        // Adjust integer value if it's negative
        if (vi & signbit) {
            vi = -(vi & mag);
        }

        // Determine column index considering sorting order
        int icol = i / nrows + 1;
        if (sortdown) {
            icol = ncols - icol + 1;
        }

        // Compute and store the result
        b[i] = static_cast<long long>(vi) + (static_cast<long long>(icol) << 32);
    }
}