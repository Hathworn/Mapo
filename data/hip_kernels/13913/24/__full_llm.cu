#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __full(int *ir, int *ic, double *data, double *od, int nrows, int ncols, int nnz) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int totalThreads = blockDim.x * gridDim.x;
    
    // Use stride loop to ensure all non-zero elements are processed efficiently.
    for (int i = id; i < nnz; i += totalThreads) {
        double v = data[i];
        int row = ir[i];
        int col = ic[i];
        od[row + col * nrows] = v;
    }
}