#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel for better performance
__global__ void __randinit(unsigned long long seed, unsigned long long offset, hiprandState *rstates) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    // Use cooperative groups to ensure proper initialization
    if (id < gridDim.x * blockDim.x) {
        hiprand_init(seed, id, offset, &rstates[id]);
    }
}