#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __randinit(unsigned long long seed, unsigned long long offset, hiprandState *rstates) {
int id = threadIdx.x + blockDim.x * blockIdx.x;
hiprand_init(seed, id, offset, &rstates[id]);
}