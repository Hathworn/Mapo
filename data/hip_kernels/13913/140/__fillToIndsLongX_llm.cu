#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __fillToIndsLongX(long long A, long long *B, long long len) {
    // Calculate global index using built-in function for better performance
    int tid = blockIdx.x * blockDim.x + threadIdx.x + 
              blockIdx.y * gridDim.x * blockDim.x;
    int step = blockDim.x * gridDim.x * gridDim.y;

    // Loop unrolling for improved performance
    for (long long i = tid; i < len; i += step) {
        B[i] = A;
    }
}