#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __transpose(float *in, int instride, float *out, int outstride, int nrows, int ncols) {
    // Calculate flattened thread index
    int bx = blockIdx.x * BLOCKDIM, by = blockIdx.y * BLOCKDIM;
    int tx = threadIdx.x, ty = threadIdx.y;
    __shared__ float tile[BLOCKDIM][BLOCKDIM+1]; // Avoid bank conflicts

    // Use a single loop over a flattened grid
    if (bx + tx < nrows) {
        int ylim = min(ncols, by + BLOCKDIM);
        for (int y = ty + by; y < ylim; y += blockDim.y) {
            tile[tx][y - by] = in[tx + bx + y * instride];
        }
    }
    __syncthreads();

    // Writing transposed data
    if (by + tx < ncols) {
        int xlim = min(nrows, bx + BLOCKDIM);
        for (int x = ty + bx; x < xlim; x += blockDim.y) {
            out[tx + by + x * outstride] = tile[x - bx][tx];
        }
    }
}