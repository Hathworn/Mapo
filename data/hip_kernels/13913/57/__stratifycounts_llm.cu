#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __stratifycounts(double *strata, int n, double *a, unsigned int *bi) {
    __shared__ unsigned int ic[SNDVALS][SNDGRPS];
    __shared__ double ss[SNDVALS];
    int istart = (int)(((long long)blockIdx.x) * n / gridDim.x);
    int iend = (int)(((long long)(blockIdx.x+1)) * n / gridDim.x);
    int bibase = SNDVALS * (blockIdx.x + istart / SBIGBLK);
    int tid = threadIdx.x + threadIdx.y * blockDim.x;

    // Load strata into shared memory with all threads in x-dimension
    if (threadIdx.y == 0 && threadIdx.x < SNDVALS) {
        ss[threadIdx.x] = strata[threadIdx.x];
    }
    __syncthreads();  // Ensure shared memory is populated

    for (int i = istart; i < iend; i += SBIGBLK) {
        // Zero-initialize ic for each SNDVALS
        if (threadIdx.y < SNDGRPS && threadIdx.x < SNDVALS) {
            ic[threadIdx.x][threadIdx.y] = 0;
        }
        __syncthreads();

        // Processing data in each thread
        for (int k = i + tid; k < min(iend, i + SBIGBLK); k += SNTHREADS) {
            double v = a[k];
            int j = 0;
            // Iterate conditionally based on strata
            #pragma unroll 8
            for (int idx = 0; idx < 8; ++idx) {
                j = (v > ss[j]) ? 2 * j + 2 : 2 * j + 1;
            }
            j = j - SNDVALS + 1;
            // Atomic increment of histogram
            atomicAdd(&ic[j][threadIdx.y], 1);
        }
        __syncthreads();

        // Reduce partial results for output
        if (threadIdx.y == 0 && threadIdx.x < SNDVALS) {
            unsigned int sum = 0;
            for (int grp = 0; grp < SNDGRPS; ++grp) {
                sum += ic[threadIdx.x][grp];
            }
            bi[bibase + threadIdx.x] = sum;
        }
        __syncthreads();
        
        // Move base index for next block
        bibase += SNDVALS;
    }
}