#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __extractmat2d(double *a, long long *b, int nrows, int ncols) {
    // Calculate unique thread index for 2D grid using blockDim.x, blockIdx.x and blockIdx.y
    int tid = threadIdx.x + blockDim.x * blockIdx.x + blockDim.x * gridDim.x * blockIdx.y;
    const int signbit = 0x80000000;
    const int mag = 0x7fffffff;
    const int totalThreads = blockDim.x * gridDim.x * gridDim.y;

    // Use tid and totalThreads for iterating over elements more efficiently
    for (int i = tid; i < nrows * ncols; i += totalThreads) {
        int vi = *((int *)&b[i]);
        if (vi & signbit) {
            vi = -(vi & mag);
        }
        a[i] = *((double *)&vi);
    }
}