#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __linfdist(float *A, int lda, float *B, int ldb, float *C, int ldc, int d, int nrows, int ncols, float p) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row < nrows && col < ncols) {  // Ensuring indices are within bounds
        float max_abs_diff = 0.0f;
        for (int i = 0; i < d; ++i) {
            float abs_diff = fabsf(A[row * lda + i] - B[i * ldb + col]);
            if (abs_diff > max_abs_diff) {
                max_abs_diff = abs_diff;  // Track maximum absolute difference
            }
        }
        C[row * ldc + col] = max_abs_diff;  // Update result matrix
    }
}