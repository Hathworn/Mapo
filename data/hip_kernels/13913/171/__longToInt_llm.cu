#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __longToInt(long long *A, int *B, int N) {
    // Calculate global thread index
    int idx = threadIdx.x + blockDim.x * blockIdx.x + blockDim.x * gridDim.x * blockIdx.y;
    
    // Optimize loop to limit unnecessary computations
    for (int i = idx; i < N; i += blockDim.x * gridDim.x * gridDim.y) {
        B[i] = static_cast<int>(A[i]);  // Use static_cast for clarity
    }
}