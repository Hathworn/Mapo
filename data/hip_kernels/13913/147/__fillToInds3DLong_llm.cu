#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __fillToInds3DLong(long long A, long long *B, int ldb, int rdb, int *I, int nrows, int *J, int ncols, int *K, int nk) {
    // Calculate flat thread index within the grid
    int tid = blockIdx.x + blockIdx.y * gridDim.x;
    tid = threadIdx.x + tid * blockDim.x;

    // Calculate total number of threads
    int numThreads = blockDim.x * gridDim.x * gridDim.y;

    int k, j, i, mapi, mapj, mapk;

    // Loop through the elements this thread is responsible for
    for (int id = tid; id < nrows * ncols * nk; id += numThreads) {
        // Calculate 3D indexes from flat index
        k = id / (nrows * ncols);
        j = (id % (nrows * ncols)) / nrows;
        i = id % nrows;

        // Map indices if necessary
        mapk = (K != NULL) ? K[k] : k;
        mapj = (J != NULL) ? J[j] : j;
        mapi = (I != NULL) ? I[i] : i;

        // Write the result
        B[mapi + ldb * (mapj + rdb * mapk)] = A;
    }
}