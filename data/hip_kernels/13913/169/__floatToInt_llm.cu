#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __floatToInt(float *A, int *B, int N) {
    int ip = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    int gridStride = blockDim.x * gridDim.x * gridDim.y; // Calculate grid stride once

    for (int i = ip; i < N; i += gridStride) {
        B[i] = static_cast<int>(A[i]); // Use static_cast for clarity
    }
}