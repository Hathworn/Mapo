#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void __extractmat2d(float *a, long long *b, int nrows, int ncols) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Flattened thread index
    int stride = blockDim.x * gridDim.x; // Total threads

    const int signbit = 0x80000000;
    const int mag = 0x7fffffff;

    for (int i = tid; i < nrows * ncols; i += stride) {
        int vi = *((int *)&b[i]);
        if (vi & signbit) {
            vi = -(vi & mag);
        }
        a[i] = *((float *)&vi);
    }
}