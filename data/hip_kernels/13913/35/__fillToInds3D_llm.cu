#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __fillToInds3D(double A, double *B, int ldb, int rdb, int *I, int nrows, int *J, int ncols, int *K, int nk) {
    // Calculate thread indices
    int ii = threadIdx.x + blockDim.x * blockIdx.x;
    int jj = threadIdx.y + blockDim.y * blockIdx.y;
    int kk = threadIdx.z + blockDim.z * blockIdx.z;
    
    // Use cache-efficient strides
    int stride_i = blockDim.x * gridDim.x;
    int stride_j = blockDim.y * gridDim.y;
    int stride_k = blockDim.z * gridDim.z;
    
    // Loop with improved memory access pattern
    for (int k = kk; k < nk; k += stride_k) {
        int mapk = (K != NULL) ? K[k] : k;
        for (int j = jj; j < ncols; j += stride_j) {
            int mapj = (J != NULL) ? J[j] : j;
            for (int i = ii; i < nrows; i += stride_i) {
                int mapi = (I != NULL) ? I[i] : i;
                B[mapi + ldb * (mapj + rdb * mapk)] = A;  // Direct access optimization
            }
        }
    }
}