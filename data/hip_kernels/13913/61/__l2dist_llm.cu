#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __l2dist(double *A, int lda, double *B, int ldb, double *C, int ldc, int d, int nrows, int ncols, double p) {
    // Calculate thread indexes
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure within bounds
    if (row < nrows && col < ncols) {
        double dist = 0.0;
        // Compute the Euclidean distance
        for (int i = 0; i < d; ++i) {
            double diff = A[row * lda + i] - B[i * ldb + col];
            dist += diff * diff;
        }
        C[row * ldc + col] = sqrt(dist);
    }
}