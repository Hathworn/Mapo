#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __fillToInds4D(float A, float *B, int ldb, int rdb, int tdb, int *I, int nrows, int *J, int ncols, int *K, int nk, int *L, int nl) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // Direct thread index
    int total_threads = gridDim.x * blockDim.x;

    int id, mapi, mapj, mapk, mapl;
    int l, k, j, i;
    
    for (id = tid; id < nrows * ncols * nk * nl; id += total_threads) {
        int tidrem = id;
        l = tidrem / (nrows * ncols * nk);  // Calculate l directly
        tidrem %= (nrows * ncols * nk);
        k = tidrem / (nrows * ncols);       // Calculate k directly
        tidrem %= (nrows * ncols);
        j = tidrem / nrows;                 // Calculate j directly
        i = tidrem % nrows;                 // Calculate i directly

        mapl = (L != NULL) ? L[l] : l;      // Use tertiary operators for compactness
        mapk = (K != NULL) ? K[k] : k;
        mapj = (J != NULL) ? J[j] : j;
        mapi = (I != NULL) ? I[i] : i;

        B[mapi + ldb * (mapj + rdb * (mapk + tdb * mapl))] = A;  // Linearize and assign
    }
}