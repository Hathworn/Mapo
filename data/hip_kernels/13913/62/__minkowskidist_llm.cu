#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __minkowskidist(double *A, int lda, double *B, int ldb, double *C, int ldc, int d, int nrows, int ncols, double p) {
    // Calculate row index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    // Calculate column index
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the row and column are within matrix bounds
    if (row < nrows && col < ncols) {
        double sum = 0.0;
        for (int k = 0; k < d; ++k) {
            // Calculate the distance increment
            double diff = A[row * lda + k] - B[k * ldb + col];
            // Accumulate the p-powered difference
            sum += pow(abs(diff), p);
        }
        // Store the p-root of the sum into result matrix C
        C[row * ldc + col] = pow(sum, 1.0 / p);
    }
}