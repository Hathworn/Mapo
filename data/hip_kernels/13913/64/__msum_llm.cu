#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __msum(double *A, int lda, double *B, int ldb, double *C, int ldc, int d, int nrows, int ncols, double p) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Compute global row index
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Compute global column index
    
    if (row < nrows && col < ncols) { // Check bounds
        double max_val = -INFINITY; // Initialize max sum to negative infinity
        for (int k = 0; k < d; ++k) { // Iterate over the dimension 'd'
            double val = A[row * lda + k] + B[k * ldb + col] - p; // Compute the value
            if (val > max_val) { // Check if it's greater than current max
                max_val = val; // Update max
            }
        }
        C[row * ldc + col] = max_val; // Store result in C
    }
}