#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __set_val(double *A, double val, int length) {
    // Calculate flattened global index within the grid
    int i = threadIdx.x + blockDim.x * blockIdx.x + blockDim.x * gridDim.x * blockIdx.y;
    // Unroll loop to increase throughput
    for (; i < length; i += blockDim.x * gridDim.x * gridDim.y) {
        A[i] = val; 
    }
}