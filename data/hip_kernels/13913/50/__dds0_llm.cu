#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __dds0(int nrows, int ncols, double *A, double *B, int *Cir, int *Cjc, double *P) {
    int row = blockIdx.x * blockDim.x + threadIdx.x; // Calculate row based on thread and block index
    if (row < nrows) { // Ensure row index is within bounds
        double sum = 0.0;
        int row_start = Cjc[row];
        int row_end = Cjc[row+1];
        for (int j = row_start; j < row_end; j++) {
            int col = Cir[j];
            sum += A[col] * B[j];
        }
        P[row] = sum;
    }
}