#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __fillToIndsX(float A, float *B, long long len) {
    // Utilize shared memory to reduce global memory access
    __shared__ float sharedA;
    if (threadIdx.x == 0) {
        sharedA = A;
    }
    __syncthreads();
    
    // Calculate the global thread index
    int tid = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
    int step = blockDim.x * gridDim.x * gridDim.y;

    // Fill array using faster shared variable
    for (long long i = tid; i < len; i += step) {
        B[i] = sharedA;
    }
}