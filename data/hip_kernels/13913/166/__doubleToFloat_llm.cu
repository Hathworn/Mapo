#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __doubleToFloat(double *A, float *B, int N) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Use stride loop for processing elements
    for (int i = idx; i < N; i += stride) {
        B[i] = (float)(A[i]);
    }
}