#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel code
__global__ void __embedmat2d(double *a, long long *b, int nrows, int ncols, int sortdown) {
    int tid = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
    const int signbit = 0x80000000;
    const int mag = 0x7fffffff;
    int total_threads = blockDim.x * gridDim.x * gridDim.y; // Pre-compute total threads

    for (int i = tid; i < nrows * ncols; i += total_threads) { // Use pre-computed total_threads
        double v = a[i];
        int vi = *((int *)&v);
        vi = (vi & signbit) ? -(vi & mag) : vi; // Simplify if-else with conditional operator

        int icol = (i / nrows + 1);
        icol = sortdown ? (ncols - icol + 1) : icol; // Use conditional operator for icol computation

        b[i] = (long long)vi + (((long long)icol) << 32);
    }
}