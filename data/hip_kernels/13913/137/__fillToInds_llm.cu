#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __fillToInds(float A, float *B, int *I, long long len) {
    // Calculate global unique thread ID
    long long tid = threadIdx.x + blockIdx.x * blockDim.x + gridDim.x * (blockIdx.y * blockDim.x);
    // Calculate the total number of threads executing the kernel
    long long step = blockDim.x * gridDim.x * gridDim.y;

    for (long long i = tid; i < len; i += step) {
        // Assign value A to specific indices in B array
        B[I[i]] = A;
    }
}