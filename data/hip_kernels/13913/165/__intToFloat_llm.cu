#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __intToFloat(int *A, float *B, int N) {
    // Calculate the global index once, instead of repeatedly inside the loop
    int idx = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
    int stride = blockDim.x * gridDim.x * gridDim.y;
    
    // Utilize stride-based looping for better data access pattern
    for (int i = idx; i < N; i += stride) {
        B[i] = static_cast<float>(A[i]);
    }
}