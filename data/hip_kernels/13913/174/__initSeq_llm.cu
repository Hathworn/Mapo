#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __initSeq(int *A, int nrows, int ncols) {
    // Calculate the position of the current thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 
              blockIdx.y * gridDim.x * blockDim.x;

    int stride = gridDim.x * gridDim.y * blockDim.x;

    // Efficient iteration over elements using stride
    for (int i = idx; i < nrows * ncols; i += stride) {
        A[i] = i % nrows;
    }
}