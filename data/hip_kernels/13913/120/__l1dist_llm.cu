#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __l1dist(float *A, int lda, float *B, int ldb, float *C, int ldc, int d, int nrows, int ncols, float p) {
    // Calculate unique indices for this thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread indexes are within matrix dimensions
    if (row < nrows && col < ncols) {
        float sum = 0.0f;
        
        // Calculate the L1 distance for the current row and column
        for (int k = 0; k < d; ++k) {
            float diff = A[row * lda + k] - B[k * ldb + col];
            sum += fabs(diff);
        }
        
        // Store the result in the output matrix C
        C[row * ldc + col] = sum;
    }
}