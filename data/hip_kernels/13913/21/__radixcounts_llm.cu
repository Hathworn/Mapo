#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __radixcounts(float *a, int n, int digit, unsigned int *bi) {
    __shared__ unsigned int ic[RNDVALS];

    int istart = blockIdx.x * n / gridDim.x;
    int iend = (blockIdx.x + 1) * n / gridDim.x;
    int tid = threadIdx.x;
    int bibase = RNDVALS * (blockIdx.x + istart / RBIGBLK);

    for (int i = istart; i < iend; i += RBIGBLK) {
        // Initialize shared memory
        ic[tid] = 0;
        __syncthreads();

        for (int j = i + tid; j < min(iend, i + RBIGBLK); j += RNTHREADS) {
            float v = a[j];
            unsigned char *cv = (unsigned char *)&v;
            // Use atomicAdd as atomicInc might not be suitable for incrementation
            atomicAdd(&ic[cv[digit]], 1);
        }
        __syncthreads();

        // Write back results from shared memory to global memory
        bi[bibase + tid] = ic[tid];
        bibase += RNDVALS;
    }
}