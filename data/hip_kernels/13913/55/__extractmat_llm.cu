#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __extractmat(double *a, int *b, long long *c, int n) {
    // Calculate a unique global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x + gridDim.x * blockIdx.y * blockDim.x;
    
    // Use local constants for better performance
    const int signbit = 0x80000000;
    const int mag = 0x7fffffff;
    
    // Use stride pattern to allow all threads to effectively iterate over data
    for (int i = tid; i < n; i += blockDim.x * gridDim.x * gridDim.y) {
        // Access elements using pointer arithmetic for efficient memory handling
        int vi = *(reinterpret_cast<int *>(&c[i]));
        if (vi & signbit) {
            vi = -(vi & mag);
        }
        a[i] = static_cast<double>(vi);  // Directly set double for clarity
        b[i] = *(reinterpret_cast<int *>(&c[i]) + 1);  // Efficient access for second part
    }
}