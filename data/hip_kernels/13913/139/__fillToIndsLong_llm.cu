#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __fillToIndsLong(long long A, long long *B, int *I, long long len) {
    // Calculate unique global thread ID in 3D grid
    int tid = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    // Calculate step size based on total number of threads
    int step = blockDim.x * gridDim.x * gridDim.y;

    // Iterate over indices for this thread
    for (long long i = tid; i < len; i += step) {
        B[I[i]] = A;  // Assign value to the specified index
    }
}