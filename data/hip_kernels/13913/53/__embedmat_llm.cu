#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __embedmat(double *a, int *b, long long *c, int n) {
    // Optimize thread index computation for readability and efficiency
    int tid = blockIdx.y * gridDim.x + blockIdx.x;
    tid = tid * blockDim.x + threadIdx.x;
    const int signbit = 0x80000000;
    const int mag = 0x7fffffff;

    // Unroll loop to increase instruction level parallelism
    for (int i = tid; i < n; i += blockDim.x * gridDim.x * gridDim.y) {
        double v = a[i];
        int vi = __double2int_rz(v); // Use HIP intrinsic for type conversion
        if (vi & signbit) {
            vi = -(vi & mag);
        }
        c[i] = (long long)vi + (((long long)b[i]) << 32);
    }
}