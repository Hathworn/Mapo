#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __msum(float *A, int lda, float *B, int ldb, float *C, int ldc, int d, int nrows, int ncols, float p) {

    // Calculate row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundaries
    if (row < nrows && col < ncols) {
        float maxValue = -FLT_MAX;
        // Accumulate maximum value
        for (int k = 0; k < d; ++k) {
            maxValue = fmaxf(maxValue, A[row * lda + k] + B[k * ldb + col]);
        }
        // Store the result multiplied by p
        C[row * ldc + col] = p * maxValue;
    }
}