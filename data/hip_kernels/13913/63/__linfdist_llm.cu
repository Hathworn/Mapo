#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __linfdist(double *A, int lda, double *B, int ldb, double *C, int ldc, int d, int nrows, int ncols, double p) {
    // Calculate global thread indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread is within matrix bounds
    if (row < nrows && col < ncols) {
        double maxDist = 0.0;
        for (int i = 0; i < d; i++) {
            // Calculate max-abs distance
            double diff = fabs(A[row * lda + i] - B[col * ldb + i]);
            if (diff > maxDist) {
                maxDist = diff;
            }
        }
        // Store the result in matrix C
        C[row * ldc + col] = maxDist;
    }
}