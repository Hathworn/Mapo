#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __poissonrnd(int n, float *A, int *B, hiprandState *rstates) {
int id = threadIdx.x + blockDim.x * blockIdx.x;
int nthreads = blockDim.x * gridDim.x;
hiprandState rstate = rstates[id];
for (int i = id; i < n; i += nthreads) {
int cr = hiprand_poisson(&rstate, A[i]);
B[i] = cr;
}
}