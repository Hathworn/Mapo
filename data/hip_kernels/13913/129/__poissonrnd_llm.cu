#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __poissonrnd(int n, float *A, int *B, hiprandState *rstates) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int nthreads = blockDim.x * gridDim.x;

    // Precompute random state once per thread
    hiprandState rstate = rstates[id];

    // Modified loop to reduce branch divergence and optimize utilization
    for (int i = id; i < n; i += nthreads) {
        if (i < n) {
            int cr = hiprand_poisson(&rstate, A[i]);
            B[i] = cr;
        }
    }

    // Save back the modified random state
    rstates[id] = rstate;
}