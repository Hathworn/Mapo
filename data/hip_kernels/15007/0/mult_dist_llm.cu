#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define H 5
#define W 5

using namespace std;

__global__ void mult_dist(int *d_A, int *d_B, int *d_C) {
    // Calculate global row and column index for A and B
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundaries to prevent out of bounds memory access
    if (i < H && j < W) {
        int Pvalue = 0;

        // Use shared memory for A and B sub-matrices
        __shared__ int sh_A[H][W];
        __shared__ int sh_B[H][W];

        // Load data into shared memory
        for (int k = 0; k < H; k += blockDim.x) {
            // Load current row of A and current column of B into shared memory
            if (k + threadIdx.x < W) {
                sh_A[i][k + threadIdx.x] = d_A[i * W + (k + threadIdx.x)];
                sh_B[k + threadIdx.y][j] = d_B[(k + threadIdx.y) * W + j];
            }
            __syncthreads(); // Synchronize to ensure all data is loaded

            // Perform partial computation
            for (int kk = 0; kk < blockDim.x && k + kk < W; ++kk) {
                Pvalue += sh_A[i][k + kk] * sh_B[k + kk][j];
            }
            __syncthreads(); // Synchronize before loading new data
        }

        // Write the computed value to the output matrix
        d_C[i * W + j] = Pvalue;
    }
}