#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use block/thread indexing to avoid global memory race conditions
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) { // Ensure only one thread performs the division
        *accuracy /= N;
    }
}