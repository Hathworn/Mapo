#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Program Parameters */
#define MAXN 15000  /* Max value of N */
#define TILE_WIDTH 32  /* Width of each block */
int N;  /* Matrix size */

/* Matrices */
float overall;
char buffer[10000];
char *pbuff;
int *classIdArray = (int *)malloc(sizeof(int)*26);
int *groupIdArray = (int *)malloc(sizeof(int)*26);
int *roomIdArray =(int *) malloc(sizeof(int)*26);
int *roomSizeAsArray = (int *)malloc(sizeof(int)*5);
int *groupSizeAsArray = (int *)malloc(sizeof(int)*11);
int *timeSlotIdArray = (int *)malloc(sizeof(int)*26);
int *profIdArray = (int *)malloc(sizeof(int)*26);
int *clashes=(int *)calloc(26*26,sizeof(int));

int   *dclassIdArray,*dgroupIdArray,*droomIdArray,*droomSizeAsArray,*dgroupSizeAsArray,*dtimeSlotIdArray,*dprofIdArray,*dclashes;

/* junk */
#define randm() 4|2[uid]&3

/* optimized kernel function */
__global__ void calculateFitness(int *classIds, int *roomIds, int *roomCapacities, int *groupIds, int *groupSizes, int *timeSlotIds, int *profIds, int *clashes)
{
    int i = threadIdx.x;
    int j = blockIdx.x;

    // Cache frequently accessed values
    int classId_i = classIds[i];
    int roomCapacity_i = roomCapacities[classId_i];
    int groupId_i = groupIds[classId_i];
    int groupSize_i = groupSizes[groupId_i];
    
    // Check for room capacity and group size
    if(classId_i == classIds[j] && roomCapacity_i < groupSize_i) {
        atomicAdd(&clashes[i*j+i], 1);  // Use atomic operation
    }
    
    // Cache shared variables to minimize global memory access
    int classId_j = classIds[j];
    int roomId_i = roomIds[classId_i];
    int roomId_j = roomIds[classId_j];
    int timeSlotId_i = timeSlotIds[classId_i];
    int timeSlotId_j = timeSlotIds[classId_j];
    int profId_j = profIds[classId_j];
    
    // Check for room and time slot conflicts
    if (roomId_i == roomId_j && timeSlotId_i == timeSlotId_j && classId_i != classId_j) {
        atomicAdd(&clashes[i*j+i], 1);  // Use atomic operation
    }
    
    // Check for professor and time slot conflicts
    if (profId_j == profIds[classId_j] && timeSlotId_i == timeSlotId_j && classId_i != classId_j) {
        atomicAdd(&clashes[i*j+i], 1);  // Use atomic operation
    }
}