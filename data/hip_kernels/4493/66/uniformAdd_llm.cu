#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) {
    __shared__ float uni;
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];
    
    __syncthreads();

    // Use blockIdx.x * blockDim.x * 2 instead of deprecated __mul24
    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x;

    // Perform the adds with conditional check for boundary
    if (address < n) {
        g_data[address] += uni;
    }
    if (address + blockDim.x < n) {
        g_data[address + blockDim.x] += uni;
    }
}