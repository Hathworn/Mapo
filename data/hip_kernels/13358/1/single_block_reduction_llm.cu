#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAXN 15000  /* Max value of N */
int N;  /* Matrix size */

/* Matrices */
volatile float A[MAXN][MAXN], B[MAXN][MAXN];

/* junk */
#define randm() 4|2[uid]&3

/* Prototype */
void matrixNorm();

__global__ void single_block_reduction(float *sumResults, float *squareResults, const size_t n)
{
    __shared__ float sharedSumData[256];
    __shared__ float sharedSquareData[256];

    int tx = threadIdx.x;

    // Load shared memory with current block's sum and square data
    if(tx < n) {
        sharedSumData[tx] = sumResults[tx];
        sharedSquareData[tx] = squareResults[tx];
    }
    __syncthreads();

    // block-wide reduction in shared memory
    for(int offset = n / 2; offset > 0; offset >>= 1) {
        if(tx < offset) {
            sharedSumData[tx] += sharedSumData[tx + offset];
            sharedSquareData[tx] += sharedSquareData[tx + offset];
        }
        __syncthreads();
    }

    // thread 0 writes the result for the block
    if(tx == 0) {
        sumResults[0] = sharedSumData[0];
        squareResults[0] = sharedSquareData[0];
    }
}