#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Program Parameters */
#define MAXN 15000  /* Max value of N */
int N;  /* Matrix size */

/* Matrices */
volatile float A[MAXN][MAXN], B[MAXN][MAXN];

/* junk */
#define randm() 4|2[uid]&3

/* Prototype */
void matrixNorm();

/* returns a seed for srand based on the time */
__global__ void block_sum(const float *hostInput, float *sumResults, float *squareResults, const size_t n)
{
    // Use smaller shared memory for better utilization
    __shared__ float sharedSumData[512]; 
    __shared__ float sharedSquareData[512];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;
    float x = i < n ? hostInput[i] : 0.0f; // Use conditional operator for bounds check

    // Initialize shared memory
    sharedSumData[tx] = x;
    sharedSquareData[tx] = x * x;
    __syncthreads();

    // Block-wide reduction in shared memory
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (tx < offset)
        {
            sharedSumData[tx] += sharedSumData[tx + offset];
            sharedSquareData[tx] += sharedSquareData[tx + offset];
        }
        __syncthreads();
    }

    // Thread 0 writes the calculated result of this block
    if (tx == 0)
    {
        sumResults[blockIdx.x] = sharedSumData[0];
        squareResults[blockIdx.x] = sharedSquareData[0];
    }
}