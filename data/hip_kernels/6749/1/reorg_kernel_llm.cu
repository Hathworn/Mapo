#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= N) return;

    // Precompute divisors
    int area = w * h;
    int channel_area = area * c;
    
    int in_index = i;
    int in_w = in_index % w;
    int in_h = (in_index / w) % h;
    int in_c = (in_index / area) % c;
    int b = in_index / channel_area;

    // Reduce the number of divisions by using multiplication
    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    int out_index = w2 + (w * stride) * (h2 + (h * stride) * (c2 + out_c * b));

    // Conditional operator to simplify if-else
    out[forward ? out_index : in_index] = x[forward ? in_index : out_index];
}