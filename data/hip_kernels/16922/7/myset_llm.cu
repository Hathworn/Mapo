#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void myset(unsigned long long *p, unsigned long long v, long long n) {
    // Calculate global thread ID
    const long long tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll loop to increase efficiency
    if (tid < n) {
        p[tid] = v;
        if (tid + blockDim.x < n) p[tid + blockDim.x] = v;
        if (tid + 2 * blockDim.x < n) p[tid + 2 * blockDim.x] = v;
        if (tid + 3 * blockDim.x < n) p[tid + 3 * blockDim.x] = v;
    }
}