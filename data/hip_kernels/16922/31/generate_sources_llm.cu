#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_sources(hiprandState *state, int n, uint32_t *verts) {
    int first = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Use one thread to check out of bounds and return early
    if (first >= n) return;

    hiprandState local_state = state[first];

    // Iterate while ensuring we do not go beyond bounds
    for (int id = first; id < n; id += stride) {
        verts[id] = hiprand(&local_state);
    }
    
    // Save the local state back to global memory
    state[first] = local_state;
}