#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_destinations(hiprandState *state, int n, const uint32_t *sources, uint32_t *destinations) {
    int first = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Cache the state of the first thread
    hiprandState local_state = state[first];
    for (int id = first; id < n; id += stride) {
        // Use modulo operation directly on n for better randomness scope
        destinations[id] = sources[hiprand(&local_state) % n];
    }

    // Save the updated state back to global memory
    state[first] = local_state;
}