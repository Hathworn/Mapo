#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multMatriz(float *da, float *db, float *dc, int num) {
    // Calculate unique global thread indices for rows and columns
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure thread operates within matrix bounds
    if (i < num && j < num) {
        float sum = 0;
        // Perform matrix multiplication for the current row and col
        for (unsigned int k = 0; k < num; k++)
            sum += da[i * num + k] * db[k * num + j];
        // Store the result in the output matrix
        dc[i*num + j] = sum;
    }
}