#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulSh(float *Md, float *Nd, float *Pd, const int WIDTH)
{
    // Allocate shared memory for tiles
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // Calculate thread indices
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

    float Pvalue = 0; // Accumulator for the compute

    for (int m = 0; m < WIDTH / TILE_WIDTH; m++) // Loop over phases
    {
        // Load tiles into shared memory
        Mds[threadIdx.y][threadIdx.x] = Md[row * WIDTH + (m * TILE_WIDTH + threadIdx.x)];
        Nds[threadIdx.y][threadIdx.x] = Nd[(m * TILE_WIDTH + threadIdx.y) * WIDTH + col];
        
        __syncthreads(); // Synchronize to ensure all tiles are loaded

        // Accumulate partial results
        for (int k = 0; k < TILE_WIDTH; k++)
        {
            Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
        }

        __syncthreads(); // Synchronize before loading new tile
    }

    // Write the result to the output matrix
    Pd[row * WIDTH + col] = Pvalue;
}