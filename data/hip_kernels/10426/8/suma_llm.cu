#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize the kernel function using shared memory
__global__ void suma(int a, int b, int *c) {
    extern __shared__ int temp[];
    int threadId = threadIdx.x;

    if (threadId == 0) {
        temp[0] = a;
        temp[1] = b;
    }
    __syncthreads();

    if (threadId == 0) {
        *c = temp[0] + temp[1];
    }
}