#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__global__ void MatrixMul(float *Md, float *Nd, float *Pd, const int WIDTH) {
    // Calculate thread ID
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

    // Boundary check to ensure valid memory access
    if (row < WIDTH && col < WIDTH) {
        float Pvalue = 0; // Accumulator for the sum
        for (int k = 0; k < WIDTH; k++) {
            Pvalue += Md[row * WIDTH + k] * Nd[k * WIDTH + col];
        }
        Pd[row * WIDTH + col] = Pvalue; // Store result
    }
}