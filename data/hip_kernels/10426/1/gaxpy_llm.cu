#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpy(double *y, double *a, double *x, int m, int n) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];

    // Check if block is within bounds
    if (bid < m) {
        // Calculate the product within threads
        if (tid < n) {
            dots_s[tid] = a[bid * n + tid] * x[tid];
        }

        __syncthreads();

        // Use a single thread to accumulate the sum
        if (tid == 0) {
            double sum = 0.0;
            for (int i = 0; i < n; i++) {
                sum += dots_s[i];
            }
            y[bid] = sum;
        }
    }
}