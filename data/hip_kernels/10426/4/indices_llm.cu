#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void indices(){
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Print conditionally to reduce overhead in large-scale executions
    if (id < blockDim.x * gridDim.x) {
        printf("blockdimy: %d  threadx: %d  Blockidx: %d  blockdimx: %d id:  %d raiz: %f\n",
               blockDim.y, threadIdx.x, blockIdx.x, blockDim.x, id, sqrt((double)id));
    }

    // __syncthreads() is unnecessary here since no shared memory is altered
}