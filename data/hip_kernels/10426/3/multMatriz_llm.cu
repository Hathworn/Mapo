#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multMatriz(float *da, float *db, float *dc, int num) {
    // Compute global row and column indices
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < num && j < num) { // Ensure thread is within matrix bounds
        float sum = 0.0f;
        for (unsigned int k = 0; k < num; k++) {
            sum += da[i * num + k] * db[k * num + j];
        }
        dc[i * num + j] = sum;
    }
}