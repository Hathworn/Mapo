#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelA(float* r, float* x, float* y, float* z, int size)
{
    // Use global thread index to handle larger data sets
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Compute stride for grid-stride loop

    // Grid-stride loop for efficient memory access and better parallelization
    for (int i = idx; i < size; i += stride) {
        r[i] = x[i] * y[i] + z[i];
    }
}