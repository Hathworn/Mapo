#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
    // Use grid stride loop for better performance on larger datasets
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < size; i += stride) {
        r[i] = x[i] * y[i] + z[i];
    }
}