#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define ARRAY_SIZE 200
#define ARRAY_BYTES ARRAY_SIZE * sizeof(float)

__global__ void CalculateSquare(float* p_out, float* p_in)
{
    // Calculate the global index for larger grids
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (index < ARRAY_SIZE) {
        float valueToSquare = p_in[index];
        p_out[index] = valueToSquare * valueToSquare;
    }
}