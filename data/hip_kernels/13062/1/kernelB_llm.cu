#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
    // Use a more efficient thread index calculation to reduce redundant operations
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for full grid coverage

    for (; i < size; i += stride) {
        r[i] = x[i] * y[i] + z[i];
    }
}