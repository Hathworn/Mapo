#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelA(float* r, float* x, float* y, float* z, int size)
{
    // Utilize global indexing for better performance.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop to cover larger array sizes through strided access.
    for (int i = idx; i < size; i += stride) {
        r[i] = x[i] * y[i] + z[i];
    }
}