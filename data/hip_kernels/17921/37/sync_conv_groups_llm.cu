#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups() {
    // Optimize using thread synchronization
    for (int i = 0; i < gridDim.x; ++i) {
        __syncthreads();  // Ensure all threads synchronize at this point
    }
}