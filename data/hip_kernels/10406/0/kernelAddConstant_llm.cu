#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

/*
* Multi-GPU sample using OpenMP for threading on the CPU side
* needs a compiler that supports OpenMP 2.0
*/

using namespace std;

// a simple kernel that simply increments each array element by b

// Optimize using 1D grid for better memory access pattern
__global__ void kernelAddConstant(int *g_a, const int b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {  // Check array bounds
        g_a[idx] += b;  // Increment array element
    }
}