#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        // Use register to load memory to reduce redundant accesses
        float value = a[idx];
        a[idx] = value * value;
    }
}