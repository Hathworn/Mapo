#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_cmp_kernel(std::size_t n, int* aptr, int* bptr, int* rptr) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Determine if the thread should perform the comparison
    if (i < n) {
        int cmp = aptr[i] < bptr[i];
        
        // Use atomic operation to ensure correctness and avoid __syncthreads_or
        if (cmp) atomicMax(rptr, 1);
    }
}