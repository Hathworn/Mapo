#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Precompute displacement values
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);
    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];
    float weight;

    // Precompute boundary check conditions
    bool withinBoundsBR = (tx < w) && (tx >= 0) && (ty < h) && (ty >= 0);
    bool withinBoundsBL = ((tx-1) < w) && ((tx-1) >= 0) && (ty < h) && (ty >= 0);
    bool withinBoundsUL = ((tx-1) < w) && ((tx-1) >= 0) && ((ty-1) < h) && ((ty-1) >= 0);
    bool withinBoundsUR = (tx < w) && (tx >= 0) && ((ty-1) < h) && ((ty-1) >= 0);

    // Fill pixel containing bottom right corner
    if (withinBoundsBR) {
        weight = dx * dy;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Fill pixel containing bottom left corner
    if (withinBoundsBL) {
        weight = (1.0f - dx) * dy;
        _atomicAdd(dst + ty * image_stride + (tx - 1), value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + (tx - 1), weight);
    }

    // Fill pixel containing upper left corner
    if (withinBoundsUL) {
        weight = (1.0f - dx) * (1.0f - dy);
        _atomicAdd(dst + (ty - 1) * image_stride + (tx - 1), value * weight);
        _atomicAdd(normalization_factor + (ty - 1) * image_stride + (tx - 1), weight);
    }

    // Fill pixel containing upper right corner
    if (withinBoundsUR) {
        weight = dx * (1.0f - dy);
        _atomicAdd(dst + (ty - 1) * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + (ty - 1) * image_stride + tx, weight);
    }
}