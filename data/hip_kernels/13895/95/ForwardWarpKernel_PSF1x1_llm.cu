#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Calculate global thread index
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Bounds check
    if (i >= h || j >= w) return;

    // Precompute row offsets
    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Load velocity components
    float u_ = __ldg(&u[flow_row_offset + j]);
    float v_ = __ldg(&v[flow_row_offset + j]);

    // Calculate corner of target pixel
    float cx = u_ * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v_ * time_scale + static_cast<float>(i) + 1.0f;

    // Convert to integer target coordinates
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Load source value and check bounds before atomic add
    float value = __ldg(&src[image_row_offset + j]);
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        // Atomic add to destination
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}
```
