#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Load loc_ into registers only once
        short2 loc = __ldg(&loc_[ptidx]); 

        // Use float variables for calculations to avoid redundant conversions
        float x_scaled = loc.x * scale; 
        float y_scaled = loc.y * scale; 
        
        // Store results
        x[ptidx] = x_scaled;
        y[ptidx] = y_scaled;
    }
}