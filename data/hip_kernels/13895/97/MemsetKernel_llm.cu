#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Compute the global thread index
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Ensure that the thread operates within matrix bounds
    if (i < h && j < w)
    {
        // Calculate linear index and set value
        const int pos = i * w + j;
        image[pos] = value;
    }
}