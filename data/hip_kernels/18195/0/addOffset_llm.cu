#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addOffset(int *dev_array, int length)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for all threads
    for (int i = tid; i < length; i += stride) // Loop over elements using stride
    {
        dev_array[i] += OFFSET;
    }
}