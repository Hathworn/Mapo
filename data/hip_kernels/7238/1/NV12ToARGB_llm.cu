#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* inference-101
*/

#define COLOR_COMPONENT_MASK            0x3FF
#define COLOR_COMPONENT_BIT_SIZE        10

#define FIXED_DECIMAL_POINT             24
#define FIXED_POINT_MULTIPLIER          1.0f
#define FIXED_COLOR_COMPONENT_MASK      0xffffffff

#define MUL(x,y)    (x*y)

__constant__ uint32_t constAlpha;
__constant__ float  constHueColorSpaceMat[9];

__device__ uint32_t RGBAPACK_10bit(float red, float green, float blue, uint32_t alpha)
{
    uint32_t ARGBpixel = 0;
    // Clamp final 10 bit results
    red   = min(max(red,   0.0f), 1023.f);
    green = min(max(green, 0.0f), 1023.f);
    blue  = min(max(blue,  0.0f), 1023.f);

    // Convert to 8 bit unsigned integers per color component
    ARGBpixel = ((((uint32_t)red   >> 2) << 24) |
                 (((uint32_t)green >> 2) << 16) |
                 (((uint32_t)blue  >> 2) <<  8) | (uint32_t)alpha);

    return ARGBpixel;
}

__device__ void YUV2RGB(uint32_t *yuvi, float *red, float *green, float *blue)
{
    const float luma = float(yuvi[0]);
    const float u    = float(yuvi[1]) - 512.0f;
    const float v    = float(yuvi[2]) - 512.0f;

    *red    = luma + 1.140f * v;
    *green  = luma - 0.395f * u - 0.581f * v;
    *blue   = luma + 2.032f * u;
}

__global__ void NV12ToARGB(uint32_t *srcImage, size_t nSourcePitch, uint32_t *dstImage, size_t nDestPitch, uint32_t width, uint32_t height)
{
    // Pre-calculate commonly used offsets/jumps to minimize repeated calculations
    const int x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t dstImagePitch = nDestPitch >> 2;
    const uint8_t *srcImageU8 = (uint8_t *)srcImage;
    const uint32_t chromaOffset = nSourcePitch * height;

    if (x >= width || y >= height) return;

    // Read 2 Luma components at a time
    uint32_t yuv101010Pel[2];
    yuv101010Pel[0] = (srcImageU8[y * nSourcePitch + x]) << 2;
    yuv101010Pel[1] = (srcImageU8[y * nSourcePitch + x + 1]) << 2;

    const int y_chroma = y >> 1;

    // Optimize chroma (Cb, Cr) interpolation to minimize branching and calculations
    uint32_t chromaCb = srcImageU8[chromaOffset + y_chroma * nSourcePitch + x];
    uint32_t chromaCr = srcImageU8[chromaOffset + y_chroma * nSourcePitch + x + 1];

    if (y & 1)  // odd scanline
    {
        if (y_chroma < ((height >> 1) - 1))
        {
            chromaCb = (chromaCb + srcImageU8[chromaOffset + (y_chroma + 1) * nSourcePitch + x] + 1) >> 1;
            chromaCr = (chromaCr + srcImageU8[chromaOffset + (y_chroma + 1) * nSourcePitch + x + 1] + 1) >> 1;
        }
    }

    // Pack chroma data
    const uint32_t chromaCbShifted = chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2);
    const uint32_t chromaCrShifted = chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2);

    yuv101010Pel[0] |= chromaCbShifted | chromaCrShifted;
    yuv101010Pel[1] |= chromaCbShifted | chromaCrShifted;

    // Perform color conversion
    uint32_t yuvi[6];
    float red[2], green[2], blue[2];

    yuvi[0] = (yuv101010Pel[0] & COLOR_COMPONENT_MASK);
    yuvi[1] = ((yuv101010Pel[0] >> COLOR_COMPONENT_BIT_SIZE) & COLOR_COMPONENT_MASK);
    yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE * 2)) & COLOR_COMPONENT_MASK);

    yuvi[3] = (yuv101010Pel[1] & COLOR_COMPONENT_MASK);
    yuvi[4] = ((yuv101010Pel[1] >> COLOR_COMPONENT_BIT_SIZE) & COLOR_COMPONENT_MASK);
    yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE * 2)) & COLOR_COMPONENT_MASK);

    YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
    YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

    // Clamp the results to RGBA
    dstImage[y * dstImagePitch + x]     = RGBAPACK_10bit(red[0], green[0], blue[0], constAlpha);
    dstImage[y * dstImagePitch + x + 1] = RGBAPACK_10bit(red[1], green[1], blue[1], constAlpha);
}