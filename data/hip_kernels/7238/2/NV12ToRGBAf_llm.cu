#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define COLOR_COMPONENT_MASK            0x3FF
#define COLOR_COMPONENT_BIT_SIZE        10

#define FIXED_DECIMAL_POINT             24
#define FIXED_POINT_MULTIPLIER          1.0f
#define FIXED_COLOR_COMPONENT_MASK      0xffffffff

#define MUL(x,y)    (x*y)

__constant__ uint32_t constAlpha;
__constant__ float  constHueColorSpaceMat[9];

__device__ void YUV2RGB(uint32_t *yuvi, float *red, float *green, float *blue)
{
    const float luma = float(yuvi[0]);
    const float u    = float(yuvi[1]) - 512.0f;
    const float v    = float(yuvi[2]) - 512.0f;

    *red    = luma + 1.140f * v;
    *green  = luma - 0.395f * u - 0.581f * v;
    *blue   = luma + 2.032f * u;
}

__global__ void NV12ToRGBAf(uint32_t* srcImage,  size_t nSourcePitch, float4* dstImage,     size_t nDestPitch, uint32_t width,       uint32_t height)
{
    int x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    uint32_t yuv101010Pel[2];
    uint32_t processingPitch = nSourcePitch;
    uint8_t *srcImageU8 = (uint8_t *)srcImage;
    
    // Optimize memory access by using local variables for frequently accessed data
    uint32_t baseYIndex = y * processingPitch;
    uint32_t baseCIndex = processingPitch * (height + (y >> 1));

    // Read 2 Luma components at a time
    yuv101010Pel[0] = (srcImageU8[baseYIndex + x    ]) << 2;
    yuv101010Pel[1] = (srcImageU8[baseYIndex + x + 1]) << 2;

    uint32_t chromaOffset = baseCIndex + x;
    uint32_t chromaCb = srcImageU8[chromaOffset];
    uint32_t chromaCr = srcImageU8[chromaOffset + 1];

    // Interpolate chroma vertically if on the odd scanline
    if (y & 1 && (y >> 1) < ((height >> 1) - 1))
    {
        chromaCb = (chromaCb + srcImageU8[chromaOffset + processingPitch] + 1) >> 1;
        chromaCr = (chromaCr + srcImageU8[chromaOffset + processingPitch + 1] + 1) >> 1;
    }

    yuv101010Pel[0] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
    yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    yuv101010Pel[1] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
    yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

    uint32_t yuvi[6];
    float red[2], green[2], blue[2];

    yuvi[0] = yuv101010Pel[0] & COLOR_COMPONENT_MASK;
    yuvi[1] = (yuv101010Pel[0] >> COLOR_COMPONENT_BIT_SIZE) & COLOR_COMPONENT_MASK;
    yuvi[2] = (yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK;

    yuvi[3] = yuv101010Pel[1] & COLOR_COMPONENT_MASK;
    yuvi[4] = (yuv101010Pel[1] >> COLOR_COMPONENT_BIT_SIZE) & COLOR_COMPONENT_MASK;
    yuvi[5] = (yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK;

    // YUV to RGB Transformation conversion
    YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
    YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

    const float s = 1.0f / 1024.0f * 255.0f;
    dstImage[y * width + x]     = make_float4(red[0] * s, green[0] * s, blue[0] * s, 1.0f);
    dstImage[y * width + x + 1] = make_float4(red[1] * s, green[1] * s, blue[1] * s, 1.0f);
}