#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_extract_roi(float* __restrict__ input, float* __restrict__ output, const char* __restrict__ mean, const int input_w, const int output_w, const int output_h, const int in_plane_r, const int in_plane_g, const int in_plane_b, const int out_plane_r, const int out_plane_g, const int out_plane_b, const int bbox_x, const int bbox_y, const int bbox_w, const int bbox_h)
{
    // Calculate output indices
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread is within bounds
    if (x < output_w && y < output_h)
    {
        // Precompute reused calculations
        float r_x = float(x) * bbox_w / output_w + bbox_x;
        float r_y = float(y) * bbox_h / output_h + bbox_y;

        float u = r_x - floor(r_x);
        float v = r_y - floor(r_y);

        int floor_rx = int(floor(r_x));
        int floor_ry = int(floor(r_y));
        int ceil_rx = int(ceil(r_x));
        int ceil_ry = int(ceil(r_y));

        // Combine pos array calculation
        int pos[4][2] = { {floor_rx, floor_ry}, {ceil_rx, floor_ry}, {floor_rx, ceil_ry}, {ceil_rx, ceil_ry} };

        // Combine map calculation
        int map[4] = { pos[0][1] * input_w + pos[0][0], pos[1][1] * input_w + pos[1][0],
                       pos[2][1] * input_w + pos[2][0], pos[3][1] * input_w + pos[3][0] };

        // Precompute weight calculations
        float s[4] = { (1-u)*(1-v), u*(1-v), (1-u)*v, u*v };

        int idx = y * output_w + x;
        // Use loop unrolling for output computation
        #pragma unroll
        for (int i = 0; i < 3; ++i) {
            int in_plane = (i == 0 ? in_plane_r : (i == 1 ? in_plane_g : in_plane_b));
            int out_plane = (i == 0 ? out_plane_r : (i == 1 ? out_plane_g : out_plane_b));
            output[idx + out_plane] = round(s[0] * input[map[0] + in_plane]
                                          + s[1] * input[map[1] + in_plane]
                                          + s[2] * input[map[2] + in_plane]
                                          + s[3] * input[map[3] + in_plane]);
        }
    }
}