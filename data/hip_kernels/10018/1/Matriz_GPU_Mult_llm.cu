#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matriz_GPU_Mult(int *a, int *b, int *c) {
    int sum = 0;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < N) {
        // Loop unrolling for performance improvement
        for (int k = 0; k < N; k += 4) {
            sum += a[j * N + k] * b[k * N + i];
            if (k + 1 < N) sum += a[j * N + k + 1] * b[(k + 1) * N + i];
            if (k + 2 < N) sum += a[j * N + k + 2] * b[(k + 2) * N + i];
            if (k + 3 < N) sum += a[j * N + k + 3] * b[(k + 3) * N + i];
        }
        c[j * N + i] = sum;
    }
}