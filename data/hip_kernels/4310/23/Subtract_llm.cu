#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Simplified computation of x index
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Simplified computation of y index
    int p = y * width + x; // Simplified computation of p
    if (x < width && y < height)
    {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // No need for __syncthreads() as there's no inter-thread dependency within the block
}