#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height) {
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    const int p_offset = y * pitch + x;
    int p = p_offset;
    
    float val = (x < width) ? d_Data[p] : d_Data[p - x];
    minvals[tx] = val;
    maxvals[tx] = val;
    
    for (int ty = 1; ty < 16; ty++, p += pitch) {
        if (x < width) {
            val = d_Data[p];
            if (val < minvals[tx]) minvals[tx] = val; // Update minimum
            if (val > maxvals[tx]) maxvals[tx] = val; // Update maximum
        }
    }
    
    __syncthreads();
    
    // Reduction to find min and max
    for (int d = 1; d < blockDim.x; d <<= 1) {
        int mask = 2 * d - 1;
        if ((tx & mask) == 0) {
            float min_pair = minvals[tx + d];
            if (min_pair < minvals[tx]) minvals[tx] = min_pair;
            
            float max_pair = maxvals[tx + d];
            if (max_pair > maxvals[tx]) maxvals[tx] = max_pair;
        }
        __syncthreads();
    }
    
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0]; // Write min result
        d_MinMax[ptr + 1] = maxvals[0]; // Write max result
    }
}