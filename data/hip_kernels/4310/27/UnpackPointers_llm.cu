#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    const int blockW = w / 16;
    int numPts = 0;

    for (int y = blockIdx.y; y < h / 32; y += gridDim.y) {    // Optimize: Use grid-stride to maximize parallelism
        for (int xBlock = blockIdx.x; xBlock < blockW; xBlock += gridDim.x) { // Optimize: Use grid-stride for x axes
            int x = xBlock * 16;                             // Calculate real x position in memory
            unsigned int val = minmax[y * w + x + tx];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}