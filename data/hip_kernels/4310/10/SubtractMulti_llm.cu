#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Calculate global indices for x, y, and z dimensions
    const int x = blockIdx.x * SUBTRACTM_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACTM_H + threadIdx.y;
    const int z = threadIdx.z;

    // Pre-compute index offsets outside the conditional check
    int sz = height * pitch;
    int p = z * sz + y * pitch + x;
    int p_offset = p + sz;

    // Check bounds before accessing data
    if (x < width && y < height && p_offset < width * height * pitch) {
        // Perform subtraction and store result
        d_Result[p] = d_Data[p] - d_Data[p_offset];
    }
}