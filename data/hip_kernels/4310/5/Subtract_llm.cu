#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    const int x = blockIdx.x * SUBTRACT_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACT_H + threadIdx.y;
    int p = y * pitch + x;
    
    // Perform bounds check before accessing global memory
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // Removed __syncthreads() as it is unnecessary here
}