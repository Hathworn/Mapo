#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    const int b = blockDim.x;
    int p = y * width + x;
  
    // Initialize min and max values
    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        minvals[tx] = FLT_MAX;
        maxvals[tx] = -FLT_MAX;
    }
    
    // Find local min and max in the y-direction
    for (int ty = 1; ty < 16; ty++) {
        p += width;
        if (x < width) {
            float val = d_Data[p];
            if (val < minvals[tx]) minvals[tx] = val;
            if (val > maxvals[tx]) maxvals[tx] = val;
        }
    }
    __syncthreads();
    
    // Perform reduction to find block-wise min and max
    for (int d = b >> 1; d > 0; d >>= 1) {
        if (tx < d) {
            if (minvals[tx + d] < minvals[tx]) minvals[tx] = minvals[tx + d];
            if (maxvals[tx + d] > maxvals[tx]) maxvals[tx] = maxvals[tx + d];
        }
        __syncthreads();
    }
    
    // Write block results to global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}