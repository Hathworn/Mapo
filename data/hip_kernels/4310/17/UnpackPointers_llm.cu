#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    // Cache thread index
    const int tx = threadIdx.x;

    // Initialize number of points for current thread
    int numPts = 0;

    // Process only necessary rows to avoid data redundancy
    for (int y = 0; y < h / 32; y++) {
        for (int x = 0; x < w; x += 16) {
            unsigned int val = minmax[y * w + x + tx];
            if (val) {
                // Unroll loop to minimize branch overhead
                #pragma unroll
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}