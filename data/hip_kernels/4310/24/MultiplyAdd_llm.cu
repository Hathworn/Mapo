#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Calculate global thread index
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate 1D index for the 2D grid
    int p = y * width + x;

    // Only compute for valid indices
    if (x < width && y < height) {
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
}