#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Initialize histogram to zero
    for (int i=0; i<13; i++)
        hist[i*32+tx] = 0.0f;
    __syncthreads();

    // Pre-compute Gaussian weights
    float i2sigma2 = -1.0f/(18.0f);
    if (tx < 15)
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));
    __syncthreads();

    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;
    int px = xp & 15;
    int x = tx - px;

    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = xp + x;
        int yi = yp + y;

        // Boundary check
        xi = max(0, min(w - 1, xi));
        yi = max(0, min(h - 1, yi));

        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = 16.0f * atan2f(dy, dx) / 3.1416f + 16.5f;
            if (bin == 32)
                bin = 0;
            float grad = sqrtf(dx * dx + dy * dy);
            hist[32 * (x - 1) + bin] += grad * gauss[x] * gauss[y];
        }
    }
    __syncthreads();

    // Reduce histogram
    for (int offset = 8; offset > 0; offset >>= 1) {
        if (tx < offset)
            hist[tx] += hist[tx + offset * 32];
        __syncthreads();
    }

    // Smoothing histogram
    if (tx < 32) {
        float v = hist[tx];
        float v_next = hist[(tx + 1) & 31];
        float v_prev = hist[(tx + 31) & 31];
        hist[32 + tx] = v;
        hist[tx] = (v > v_next && v >= v_prev) ? v : 0.0f;
    }
    __syncthreads();

    // Find peaks in histogram
    if (tx == 0) {
        float maxval1 = 0.0f, maxval2 = 0.0f;
        int i1 = -1, i2 = -1;
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }
        // Sub-pixel peak refinement
        float val1 = hist[32 + ((i1 + 1) & 31)];
        float val2 = hist[32 + ((i1 + 31) & 31)];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Orient[bx] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);

        if (maxval2 < 0.8f * maxval1) i2 = -1;
        if (i2 >= 0) {
            val1 = hist[32 + ((i2 + 1) & 31)];
            val2 = hist[32 + ((i2 + 31) & 31)];
            float peak2 = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Orient[bx + maxPts] = 11.25f * (peak2 < 0.0f ? peak2 + 32.0f : peak2);
        } else {
            d_Orient[bx + maxPts] = i2;
        }
    }
}