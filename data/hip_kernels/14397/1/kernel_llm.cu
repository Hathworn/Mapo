#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Calculate total stride once

    for (int i = tid; i < n; i += stride) {
        float value = pow(3.14159, i); // Calculate pow once
        x[i] = sqrt(value); // Store sqrt of value
    }
}