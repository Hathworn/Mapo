#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define NUM_THREADS 512

__global__ void opt_cond_itr(int num_train_cases, double *opt_cond, double alpha_high, double alpha_high_prev, int high_label, int high_indx, double alpha_low, double alpha_low_prev, int low_label, int low_indx, double *kernel_val_mat) {
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Precompute constant terms outside the if-statement for efficiency
    double delta_alpha_high = (alpha_high - alpha_high_prev) * high_label;
    double delta_alpha_low = (alpha_low - alpha_low_prev) * low_label;

    if(global_id < num_train_cases) {
        double high_value = kernel_val_mat[high_indx * num_train_cases + global_id];
        double low_value = kernel_val_mat[low_indx * num_train_cases + global_id];
        
        // Simplified update formula using precomputed values
        opt_cond[global_id] += delta_alpha_high * high_value + delta_alpha_low * low_value;
    }
}