#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel function to use shared memory
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID

    // Use shared memory for better memory access pattern
    __shared__ double shared_a[256];  // Adjust size as per maximum blockDim.x
    __shared__ double shared_b[256];  // Adjust size as per maximum blockDim.x

    if (id < n) {
        shared_a[threadIdx.x] = a[id]; // Load elements to shared memory
        shared_b[threadIdx.x] = b[id]; // Load elements to shared memory
        __syncthreads();               // Ensure all threads have loaded

        c[id] = shared_a[threadIdx.x] + shared_b[threadIdx.x]; // Perform computation
    }
}