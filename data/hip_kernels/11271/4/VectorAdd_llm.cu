#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index is within bounds
    if (i < n) {
        // Perform vector addition
        c[i] = a[i] + b[i];
    }
}