#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index is within bounds
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}