#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int n, float *x, float *y)
{
    // Compute global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for grid-strided loop

    // Use grid-stride loop for efficient memory access and more workload distribution
    for (; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
    }
}