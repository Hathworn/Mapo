#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
    // Calculate once to reduce redundant calculations
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds and perform addition
    if (globalThreadId < n)
        c[globalThreadId] = a[globalThreadId] + b[globalThreadId];
}