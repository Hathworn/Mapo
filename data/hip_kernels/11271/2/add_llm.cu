#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int n, float *x, float *y)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop to handle large n
    int stride = gridDim.x * blockDim.x;
    for (int index = i; index < n; index += stride)
    {
        y[index] = x[index] + y[index];
    }
}