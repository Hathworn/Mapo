#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized CUDA Kernel function to add the elements of two arrays on the GPU

__global__ void add(int n, float *x, float *y)
{
    // Calculate global index for each thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use global index and stride to process full data range
    for (int i = index; i < n; i += stride)
    {
        y[i] = x[i] + y[i];
    }
}