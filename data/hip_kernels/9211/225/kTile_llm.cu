#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Cache total number of threads
    const int numThreads = blockDim.x * gridDim.x;

    // Iterate over global thread stride
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Compute target position
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        
        // Compute source position using modulo to create tiling effect
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        
        // Assign from source to target, using computed positions
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}