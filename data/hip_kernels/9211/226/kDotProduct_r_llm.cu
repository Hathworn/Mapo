#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;

    // Initialize shared memory
    float sum = 0.0f;
    if (eidx < numElements) {
        // Loop unrolling for coalesced memory access
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            sum += a[eidx] * b[eidx];
        }
    }
    // Store computed sum in shared memory
    shmem[threadIdx.x] = sum;

    // Reduce within shared memory using threads
    __syncthreads();
    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    __syncthreads();
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    // Write result from the first thread of the block
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}