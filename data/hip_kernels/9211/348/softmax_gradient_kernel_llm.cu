#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SOFTMAX_NUM_THREADS 256

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // Unrolled loop to improve memory access pattern
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Faster reduction using warp-level reduction
    if (idx < 32) {
        for (int i = idx + 32; i < blockDim.x; i += 32) {
            reduction_buffer[idx] += reduction_buffer[i];
        }
    }
    __syncthreads();

    // The reduction now only needs to be performed by one warp
    if (idx == 0) {
        tmp = 0.0f;
        for (int i = 0; i < 32; ++i) {
            tmp += reduction_buffer[i];
        }
        reduction_buffer[0] = tmp;
    }
    __syncthreads();

    // Compute gradient with reduced memory latency
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}