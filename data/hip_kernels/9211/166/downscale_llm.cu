#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii /= d3;
    z = ii % d2;
    ii /= d2;
    y = ii % d1;
    ii /= d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii /= d3;
    z = ii % d2;
    ii /= d2;
    y = ii % d1;
    ii /= d1;
    x = ii;
    w /= scale_factor;
    z /= scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = blockIdx.y * (blockDim.y * gridDim.x) * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x + threadIdx.y * blockDim.x;  // Optimize block and thread index calculation
    if (ii >= no_elements) return;
    int scale_factor_sq = scale_factor * scale_factor;  // Precompute scale factor squared
    int base_ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, 0, 0); // Base index computation
    for (int off = 0; off < scale_factor_sq; off++) {
        int i = off / scale_factor;  // Precompute x offset
        int j = off % scale_factor;  // Precompute y offset
        int ipidx = base_ipidx + translate_idx_inv(0, 1, d2, d3, scale_factor, i, j);  // Reuse base_ipidx and increment with offsets
        gradInput_data[ii] += gradOutput_data[ipidx];
    }
}