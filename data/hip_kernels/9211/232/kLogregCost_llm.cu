#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Use ternary operator for concise conditional assignment
        correctProbs[tx] = (labelp != maxp) ? 0 : 1.0f / float([&]{
            int numMax = 0;
            #pragma unroll // Improve loop performance
            for (int i = 0; i < numOut; i++) {
                numMax += probs[i * numCases + tx] == maxp;
            }
            return numMax;
        }());
    }
}