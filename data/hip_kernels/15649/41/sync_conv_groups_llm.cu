#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups() {
    // Use dynamic thread and block synchronization
    __syncthreads(); // Ensure all threads in a block are synchronized
    __threadfence(); // Ensure memory visibility after synchronization
}