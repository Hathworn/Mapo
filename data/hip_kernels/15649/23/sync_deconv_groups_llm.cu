#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_deconv_groups() {
    // Unified synchronization call for all threads in a block
    __syncthreads();
}