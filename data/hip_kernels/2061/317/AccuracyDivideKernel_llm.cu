#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load from global to register for faster access
    float acc = *accuracy;

    // Perform division
    acc /= N;

    // Write result back to global memory
    *accuracy = acc;
}