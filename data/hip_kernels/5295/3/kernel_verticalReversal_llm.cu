#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_verticalReversal(dev_t *src, dev_t *dst, uint pitch_src, uint pitch_dst, uint pixel_w, uint pixel_h)
{
    // Calculate global thread indices
    unsigned int dim_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int dim_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread is within image bounds
    if (dim_x < pixel_w && dim_y < pixel_h) {
        // Read and write using coalesced memory access
        size_t index_src = dim_y * pitch_src / 4 + dim_x;
        size_t index_dst = (pixel_h - 1 - dim_y) * pitch_dst / 4 + dim_x;
        uint32_t rgba = *((uint32_t*)src + index_src);
        *((uint32_t*)dst + index_dst) = rgba;
    }
}