#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ u_char clamp(float t)
{
    if (t < 0) {
        return 0;
    } else if (t > 255) {
        return 255;
    }
    return t;
}

__global__ void kernel_colorSpaceYUV420PToRGBA(dev_t *src, dev_t *dst, int pitch_src, int pitch_dst, int w, int h)
{
    // Calculate global thread positions
    unsigned int dim_x = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int dim_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (dim_x < w && dim_y < h) {
        // Consolidate and simplify memory access
        u_char *src_ptr = (u_char*)src;
        int y_index = dim_y * pitch_src + dim_x;
        int uv_offset = h * pitch_src;
        int u_index = (h + dim_y / 4) * pitch_src + dim_x / 2;
        int v_index = (h * 5 + dim_y) / 4 * pitch_src + dim_x / 2;

        int y = src_ptr[y_index];
        int u = src_ptr[u_index];
        int v = src_ptr[v_index];

        // Compute RGB values using simplified arithmetic
        int r = clamp(y + 1.402f * (v - 128) + 0.5f);
        int g = clamp(y - 0.34414f * (u - 128) - 0.71414f * (v - 128) + 0.5f);
        int b = clamp(y + 1.772f * (u - 128) + 0.5f);

        // Optimize memory writes by grouping writes together
        u_char *dst_ptr = (u_char*)dst;
        int dst_index = dim_y * pitch_dst + dim_x * 4;
        dst_ptr[dst_index] = r;
        dst_ptr[dst_index + 1] = g;
        dst_ptr[dst_index + 2] = b;
        dst_ptr[dst_index + 3] = 255;  // Alpha channel set to max
    }
}