#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "kernel_renderRGBA2Surface_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    hipSurfaceObject_t surface = 1;
dev_t *src = NULL;
hipMalloc(&src, XSIZE*YSIZE);
int pitch = 2;
int pixel_w = 1;
int pixel_h = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
kernel_renderRGBA2Surface<<<gridBlock, threadBlock>>>(surface,src,pitch,pixel_w,pixel_h);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
kernel_renderRGBA2Surface<<<gridBlock, threadBlock>>>(surface,src,pitch,pixel_w,pixel_h);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
kernel_renderRGBA2Surface<<<gridBlock, threadBlock>>>(surface,src,pitch,pixel_w,pixel_h);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}