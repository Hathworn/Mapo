#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_horizontalReversal(dev_t *src, dev_t *dst, uint pitch_src, uint pitch_dst, uint pixel_w, uint pixel_h)
{
    // Calculate the global index in x and y directions
    unsigned int dim_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int dim_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread is within bounds and perform reversal only when in bounds
    if (dim_x < pixel_w && dim_y < pixel_h) {
        // Load source pixel data
        uint32_t rgba = *((uint32_t*)((char*)src + dim_y * pitch_src) + dim_x);
        // Reverse and store in destination
        *((uint32_t*)((char*)dst + dim_y * pitch_dst) + (pixel_w - 1 - dim_x)) = rgba;
    }
}