#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ u_char clamp(float t) {
    if (t < 0) {
        return 0;
    } else if (t > 255) {
        return 255;
    }
    return t;
}

__global__ void kernel_colorSpaceRGBAToYUV420P(dev_t *src, dev_t *dst, int pitch_src, int pitch_dst, int w, int h) {
    unsigned int dim_x = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int dim_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (dim_x < w && dim_y < h) {
        // Load and unpack the RGBA value
        uint32_t rgba = *((uint32_t*)((u_char*)dst + dim_y * pitch_dst) + dim_x);
        int r = rgba >> 24;
        int g = (rgba >> 16) & 0xff;
        int b = (rgba >> 8) & 0xff;

        // Cache computed clamp values and write YUV values
        u_char y_value = clamp(0.299f * r + 0.587f * g + 0.114f * b);
        *((u_char*)((u_char*)src + dim_y * pitch_src) + dim_x) = y_value;

        // Re-use calculations and avoid repetition
        if (dim_x % 2 == 0 && dim_y % 2 == 0) {
            u_char u_value = clamp(-0.1687f * r - 0.3313f * g + 0.5f * b + 128);
            u_char v_value = clamp(0.5f * r - 0.4187f * g - 0.0813f * b + 128);
            *((u_char*)(src + (h + dim_y / 4) * pitch_src) + dim_x / 2) = u_value;
            *((u_char*)(src + (h * 5 + dim_y) / 4 * pitch_src) + dim_x / 2) = v_value;
        }
    }
}