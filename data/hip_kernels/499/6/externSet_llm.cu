#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void externSet(int* variablesMem, int* lastValuesMem, int nQueen, int nVariableCollection) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute strides to reduce repetitive calculations
    int totalVariables = nVariableCollection * nQueen * nQueen;
    int collectionLimit = nVariableCollection * nQueen;

    if (index < totalVariables) {
        variablesMem[index] = 1;

        // Optimize branching by checking bounds once
        if (index < collectionLimit) {
            lastValuesMem[index] = 0;
        }
    }
}