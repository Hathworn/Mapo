#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void externUndoAssignParallel(int* domain, int size, int value) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Check if index is valid and not equal to value
    if (idx < size && idx != value) {
        // Use atomic operation to increment domain element to avoid race conditions
        atomicAdd(&domain[idx], 1);
    }
}