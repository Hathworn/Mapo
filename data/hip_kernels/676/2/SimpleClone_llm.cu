#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SimpleClone(const float *background, const float *target, const float *mask, float *output, const int wb, const int hb, const int wt, const int ht, const int oy, const int ox)
{
    // Calculate thread's target coordinates
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds for target region
    if (yt < ht && xt < wt) {
        const int curt = wt * yt + xt;

        // Proceed only if mask condition is met
        if (mask[curt] > 127.0f) {
            const int yb = oy + yt, xb = ox + xt;

            // Check bounds for background region
            if (yb < hb && xb < wb && yb >= 0 && xb >= 0) {
                const int curb = wb * yb + xb;

                // Perform copy of RGB values
                output[curb * 3 + 0] = target[curt * 3 + 0];
                output[curb * 3 + 1] = target[curt * 3 + 1];
                output[curb * 3 + 2] = target[curt * 3 + 2];
            }
        }
    }
}