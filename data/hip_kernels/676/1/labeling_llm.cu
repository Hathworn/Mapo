#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void labeling(const char *text, int *pos, int text_size) {
    // Calculate the global index for the current thread
    int index = blockDim.x * blockIdx.x + threadIdx.x; 

    // Check if the index is within bounds
    if (index >= text_size) {
        return;
    }

    // Initialize the position to zero
    pos[index] = 0;

    // Check if the current text character is a space or non-printable
    if (text[index] <= ' ') {
        return;
    }

    // Calculate distance to the last non-space character
    for (int k = index - 1; k >= 0; k--) {
        if (text[k] <= ' ') {
            pos[index] = index - k;
            return;
        }
    }

    // If no space before, the position is index + 1
    pos[index] = index + 1;
}