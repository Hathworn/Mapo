#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void assign_add(float *target, const float *source) {
    target[0] += source[0];
    target[1] += source[1];
    target[2] += source[2];
}

__global__ void PoissonImageCloningIteration(const float *fixed, const float *mask, const float *source, float *target, const int wt, const int ht) {
    // Compute unique thread index
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Guard against accessing out-of-bound memory
    if (yt >= ht || xt >= wt) return;
    
    const int curt = wt * yt + xt;
    const int Nt = curt - wt;
    const int Wt = curt - 1;
    const int St = curt + wt;
    const int Et = curt + 1;
    
    float sum[3] = {};
    
    // Accumulate fixed vector
    assign_add(sum, &fixed[curt * 3]);
    
    // Accumulate source vectors selectively based on mask and bounds checking
    if (yt > 0 && mask[Nt] > 127.0f) {
        assign_add(sum, &source[Nt * 3]);
    }
    if (xt > 0 && mask[Wt] > 127.0f) {
        assign_add(sum, &source[Wt * 3]);
    }
    if (yt < ht-1 && mask[St] > 127.0f) {
        assign_add(sum, &source[St * 3]);
    }
    if (xt < wt-1 && mask[Et] > 127.0f) {
        assign_add(sum, &source[Et * 3]);
    }
    
    // Store computed average into target
    target[curt * 3 + 0] = sum[0] / 4;
    target[curt * 3 + 1] = sum[1] / 4;
    target[curt * 3 + 2] = sum[2] / 4;
}