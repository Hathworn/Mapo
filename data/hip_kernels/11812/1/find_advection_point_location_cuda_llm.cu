#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int locationAlgo(double *x, double xadv, unsigned int nx)
{
    unsigned int location = 0;
    while (x[location] < xadv && location < nx)
        location++;
    if (location == 0)
        return location;
    else
        return location - 1;
}

__global__ void find_advection_point_location_cuda(double *x, double *y, double *xadv, double *yadv, unsigned int nx, unsigned int ny, unsigned int *cellx, unsigned int *celly, unsigned int *tracker, double xlim1, double xlim2, double ylim1, double ylim2, unsigned int TileSize)
{
    unsigned int index_x = blockIdx.x * TileSize + threadIdx.x;
    unsigned int index_y = blockIdx.y * TileSize + threadIdx.y;
    unsigned int indexToWrite = index_y * nx + index_x;

    bool xoutofbounds = !(xadv[indexToWrite] > xlim1 && xadv[indexToWrite] < xlim2);
    bool youtofbounds = !(yadv[indexToWrite] > ylim1 && yadv[indexToWrite] < ylim2);

    if (!xoutofbounds && !youtofbounds) {
        tracker[indexToWrite] = 1;
        cellx[indexToWrite] = locationAlgo(x, xadv[indexToWrite], nx);
        celly[indexToWrite] = locationAlgo(y, yadv[indexToWrite], ny);
    } else if (!xoutofbounds && youtofbounds) {
        tracker[indexToWrite] = 2;
        cellx[indexToWrite] = locationAlgo(x, xadv[indexToWrite], nx);
        celly[indexToWrite] = (yadv[indexToWrite] <= ylim1) ? 0 : (ny - 2);
    } else if (xoutofbounds && !youtofbounds) {
        tracker[indexToWrite] = 3;
        celly[indexToWrite] = locationAlgo(y, yadv[indexToWrite], ny);
        cellx[indexToWrite] = (xadv[indexToWrite] <= xlim1) ? 0 : (nx - 2);
    } else if (xoutofbounds && youtofbounds) {
        tracker[indexToWrite] = 4;
    }
}