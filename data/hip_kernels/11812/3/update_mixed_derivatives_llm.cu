#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_mixed_derivatives(double *temppsix, double *temppsiy, double *temppsixy, unsigned int nx, unsigned int ny, double dx, double dy, unsigned int TileSize)
{
    // Calculate block and thread indices
    unsigned int bx = blockIdx.x;
    unsigned int by = blockIdx.y;
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    
    // Calculate global indices for accessing the arrays
    unsigned int index_x = bx * TileSize + tx;
    unsigned int index_y = by * TileSize + ty;
    unsigned int indexToWrite = index_y * nx + index_x;

    // Ensure indices are within bounds
    if (index_x < nx && index_y < ny) {
        if ((index_y == 0 || index_y == ny - 1) && (index_x != 0 && index_x != nx - 1)) {
            // Boundary conditions at top and bottom edges
            temppsixy[indexToWrite] = (temppsiy[indexToWrite+1] - temppsiy[indexToWrite-1]) / (2.0 * dx);
        } else if ((index_y != 0 && index_y != ny - 1) && (index_x == 0 || index_x == nx - 1)) {
            // Boundary conditions at left and right edges
            temppsixy[indexToWrite] = (temppsix[indexToWrite + nx] - temppsix[indexToWrite - nx]) / (2.0 * dy);
        } else if ((index_y == 0 || index_y == ny - 1) && (index_x == 0 || index_x == nx - 1)) {
            // Corner cases
            if (index_y == 0 && index_x == 0) {
                double d1 = (temppsiy[1] - temppsiy[0]) / dx;
                double d2 = (temppsix[nx] - temppsix[0]) / dy;
                double d3 = (temppsix[nx+1] - temppsix[1]) / dy;
                double d4 = (temppsiy[nx+1] - temppsiy[nx]) / dx;
                temppsixy[indexToWrite] = 0.75 * (d1 + d2) - 0.25 * (d3 + d4);
            } else if (index_y == 0 && index_x == nx - 1) {
                double d1 = (temppsiy[nx-1] - temppsiy[nx-2]) / dx;
                double d2 = (temppsix[nx+nx-2] - temppsix[nx-2]) / dy;
                double d3 = (temppsix[nx+nx-1] - temppsix[nx-1]) / dy;
                double d4 = (temppsiy[nx+nx-1] - temppsiy[nx+nx-2]) / dx;
                temppsixy[indexToWrite] = 0.75 * (d1 + d3) - 0.25 * (d2 + d4);
            } else if (index_y == ny-1 && index_x == 0) {
                double d1 = (temppsiy[nx*(ny-2)+1] - temppsiy[nx*(ny-2)]) / dx;
                double d2 = (temppsix[nx*(ny-1)] - temppsix[nx*(ny-2)]) / dy;
                double d3 = (temppsix[nx*(ny-1)] - temppsix[nx*(ny-2)+1]) / dy;
                double d4 = (temppsiy[nx*(ny-1)+1] - temppsiy[nx*(ny-1)]) / dx;
                temppsixy[indexToWrite] = 0.75 * (d2 + d4) - 0.25 * (d3 + d1);
            } else if (index_y == ny-1 && index_x == nx-1) {
                double d1 = (temppsiy[nx*(ny-2)+nx-1] - temppsiy[nx*(ny-2)+nx-2]) / dx;
                double d2 = (temppsix[nx*(ny-1)+nx-2] - temppsix[nx*(ny-2)+nx-2]) / dy;
                double d3 = (temppsix[nx*(ny-1)+nx-1] - temppsix[nx*(ny-2)+nx-1]) / dy;
                double d4 = (temppsiy[nx*(ny-1)+nx-1] - temppsiy[nx*(ny-1)+nx-2]) / dx;
                temppsixy[indexToWrite] = 0.75 * (d3 + d4) - 0.25 * (d1 + d2);
            }
        } else {
            // General case
            double dxy1 = (temppsiy[indexToWrite+1] - temppsiy[indexToWrite-1]) / (2.0 * dx);
            double dxy2 = (temppsix[indexToWrite + nx] - temppsix[indexToWrite - nx]) / (2.0 * dy);
            temppsixy[indexToWrite] = (dxy1 + dxy2) * 0.5;
        }
    }
}