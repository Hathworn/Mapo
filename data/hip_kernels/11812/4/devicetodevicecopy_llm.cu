#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void devicetodevicecopy(double *dphi, double *dpsix, double *dpsiy, double *mphi, double *mpsix, double *mpsiy, unsigned int nx, unsigned int TileSize)
{
    // Use shared memory for optimized memory access
    __shared__ double shared_dphi[1024];
    __shared__ double shared_dpsix[1024];
    __shared__ double shared_dpsiy[1024];
  
    unsigned int bx = blockIdx.x;
    unsigned int by = blockIdx.y;
  
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
  
    unsigned int index_x = bx * TileSize + tx;
    unsigned int index_y = by * TileSize + ty;
    unsigned int indexInBlock = ty * TileSize + tx;
  
    unsigned int indexToWrite = index_y * nx + index_x;

    // Load data into shared memory
    shared_dphi[indexInBlock] = dphi[indexToWrite];
    shared_dpsix[indexInBlock] = dpsix[indexToWrite];
    shared_dpsiy[indexInBlock] = dpsiy[indexToWrite];

    __syncthreads(); // Ensure all threads have loaded the data
    
    // Write data from shared memory to global memory
    mphi[indexToWrite] = shared_dphi[indexInBlock];
    mpsix[indexToWrite] = shared_dpsix[indexInBlock];
    mpsiy[indexToWrite] = shared_dpsiy[indexInBlock];
}