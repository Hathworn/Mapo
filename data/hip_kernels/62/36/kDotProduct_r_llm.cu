#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numCols, const uint numElements) {
    extern __shared__ float shmem[]; // Use dynamic shared memory

    uint eidx = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    // Loop unrolling for efficiency
    for (uint idx = eidx; idx < numElements; idx += numCols) {
        sum += a[idx] * b[idx];
    }

    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Efficient reduction loop
    for (uint s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            shmem[threadIdx.x] += shmem[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}