#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Utilize blockDim to calculate thread index directly for improved readability
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    /* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(seed, tidx, 0, &state[tidx]);
}