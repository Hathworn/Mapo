#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    const unsigned int numEls = tgtWidth * tgtHeight;

    // Use shared memory for better memory access patterns
    __shared__ float sharedSrc[32 * 32]; // Example size, adjust according to actual usage

    for (uint i = idx; i < numEls; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Load elements into shared memory
        if (srcY * srcWidth + srcX < srcWidth * srcHeight) {
            sharedSrc[threadIdx.x] = src[srcY * srcWidth + srcX];
            __syncthreads(); // Synchronize to ensure all threads have loaded their elements
            tgt[i] = sharedSrc[threadIdx.x];
        }
    }
}