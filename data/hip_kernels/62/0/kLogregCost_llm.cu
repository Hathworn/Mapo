#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LOGREG_ERR_THREADS_X 256 // Assuming this macro is defined elsewhere

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Optimize conditional logic to simplify the calculation of correctProbs
        float correctProb = 0;
        if (labelp == maxp) {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp);
            }
            correctProb = 1.0f / float(numMax);
        }
        correctProbs[tx] = correctProb;
    }
}