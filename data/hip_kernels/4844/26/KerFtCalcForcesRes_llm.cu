#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerFtCalcForcesRes(unsigned ftcount, bool simulate2d, double dt, const float3 *ftoomega, const float3 *ftovel, const double3 *ftocenter, const float3 *ftoforces, float3 *ftoforcesres, double3 *ftocenterres) {
    const unsigned cf = blockIdx.x * blockDim.x + threadIdx.x;
    if (cf < ftcount) {
        // Load inputs from global memory once.
        float3 fomega = ftoomega[cf];
        float3 fvel = ftovel[cf];
        float3 face = ftoforces[cf * 2];
        float3 omegaace = ftoforces[cf * 2 + 1];
        double3 fcenter = ftocenter[cf];

        // Update omega using FMA for better precision and performance.
        fomega.x = fma(dt, omegaace.x, fomega.x);
        fomega.y = fma(dt, omegaace.y, fomega.y);
        fomega.z = fma(dt, omegaace.z, fomega.z);

        if (simulate2d) {
            // Zero out specific components for 2D simulation.
            face.y = 0;
            fomega.x = 0;
            fomega.z = 0;
            fvel.y = 0;
        }

        // Update center using FMA.
        fcenter.x = fma(dt, fvel.x, fcenter.x);
        fcenter.y = fma(dt, fvel.y, fcenter.y);
        fcenter.z = fma(dt, fvel.z, fcenter.z);

        // Update velocity using FMA.
        fvel.x = fma(dt, face.x, fvel.x);
        fvel.y = fma(dt, face.y, fvel.y);
        fvel.z = fma(dt, face.z, fvel.z);

        // Store the updated results back to global memory.
        ftoforcesres[cf * 2] = fomega;
        ftoforcesres[cf * 2 + 1] = fvel;
        ftocenterres[cf] = fcenter;
    }
}