#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KerSortDataParticles(unsigned n, unsigned pini, const unsigned *sortpart, const float4 *a, float4 *a2)
{
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x; // Particle index
    if (p < n) {
        unsigned oldpos = (p < pini ? p : sortpart[p]);
        a2[p] = a[oldpos]; // Assign sorted data
    }
}