#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "KerSortDataParticles_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    unsigned n = XSIZE*YSIZE;
unsigned pini = 1;
const unsigned *sortpart = NULL;
hipMalloc(&sortpart, XSIZE*YSIZE);
const float4 *a = NULL;
hipMalloc(&a, XSIZE*YSIZE);
float4 *a2 = NULL;
hipMalloc(&a2, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
KerSortDataParticles<<<gridBlock, threadBlock>>>(n,pini,sortpart,a,a2);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
KerSortDataParticles<<<gridBlock, threadBlock>>>(n,pini,sortpart,a,a2);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
KerSortDataParticles<<<gridBlock, threadBlock>>>(n,pini,sortpart,a,a2);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}