#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerPeriodicDuplicateNormals(unsigned n, unsigned pini, const unsigned *listp, float3 *normals, float3 *motionvel)
{
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < n) {
        unsigned pnew = p + pini;
        unsigned rp = listp[p];
        unsigned pcopy = (rp & 0x7FFFFFFF);

        // Avoid recalculating 'pcopy' when copying normals
        float3 normal = normals[pcopy];
        normals[pnew] = normal;

        if (motionvel) {
            float3 velocity = motionvel[pcopy];
            motionvel[pnew] = velocity;
        }
    }
}