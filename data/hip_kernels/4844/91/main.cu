#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "KerPeriodicDuplicateNormals.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    unsigned n = XSIZE*YSIZE;
unsigned pini = 1;
const unsigned *listp = NULL;
hipMalloc(&listp, XSIZE*YSIZE);
float3 *normals = NULL;
hipMalloc(&normals, XSIZE*YSIZE);
float3 *motionvel = NULL;
hipMalloc(&motionvel, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
KerPeriodicDuplicateNormals<<<gridBlock, threadBlock>>>(n,pini,listp,normals,motionvel);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
KerPeriodicDuplicateNormals<<<gridBlock, threadBlock>>>(n,pini,listp,normals,motionvel);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
KerPeriodicDuplicateNormals<<<gridBlock, threadBlock>>>(n,pini,listp,normals,motionvel);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}