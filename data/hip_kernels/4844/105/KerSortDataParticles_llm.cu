#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel function for sorting data particles
__global__ void KerSortDataParticles(unsigned n, unsigned pini, const unsigned *sortpart, const float4 *a, float4 *a2)
{
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x; // Global particle index
    if (p < n) {
        unsigned oldpos = (p < pini) ? p : sortpart[p]; // Calculate old position
        a2[p] = __ldg(&a[oldpos]); // Use __ldg for read-only cache load
    }
}