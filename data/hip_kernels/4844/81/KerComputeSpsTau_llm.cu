#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerComputeSpsTau(unsigned n, unsigned pini, float smag, float blin, const float4 *velrhop, const float2 *gradvelff, float2 *tauff) {
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < n) {
        const unsigned p1 = p + pini;
        // Load gradients into registers
        float2 rr0 = gradvelff[p1 * 3];
        float2 rr1 = gradvelff[p1 * 3 + 1];
        float2 rr2 = gradvelff[p1 * 3 + 2];

        // Unroll constants
        const float grad_xx = rr0.x, grad_xy = rr0.y;
        const float grad_xz = rr1.x, grad_yy = rr1.y;
        const float grad_yz = rr2.x, grad_zz = rr2.y;
        
        // Reduce computation duplications
        const float gradxx_yy_zz = grad_xx * grad_xx + grad_yy * grad_yy + grad_zz * grad_zz;
        const float prr = grad_xy * grad_xy + grad_xz * grad_xz + grad_yz * grad_yz + 2.0f * gradxx_yy_zz;
        
        const float visc_sps = smag * rsqrtf(prr);
        const float div_u = grad_xx + grad_yy + grad_zz;
        const float sps_k = (2.0f / 3.0f) * visc_sps * div_u;
        const float sps_blin = blin * prr;
        const float sumsps = -(sps_k + sps_blin);
        
        // Pre-compute reused expressions
        const float twovisc_sps = 2.0f * visc_sps;
        float one_rho2 = 1.0f / velrhop[p1].w;

        // Compute new tau values
        const float tau_xx = one_rho2 * (twovisc_sps * grad_xx + sumsps);
        const float tau_xy = one_rho2 * (visc_sps * grad_xy);
        tauff[p1 * 3] = make_float2(tau_xx, tau_xy);

        const float tau_xz = one_rho2 * (visc_sps * grad_xz);
        const float tau_yy = one_rho2 * (twovisc_sps * grad_yy + sumsps);
        tauff[p1 * 3 + 1] = make_float2(tau_xz, tau_yy);

        const float tau_yz = one_rho2 * (visc_sps * grad_yz);
        const float tau_zz = one_rho2 * (twovisc_sps * grad_zz + sumsps);
        tauff[p1 * 3 + 2] = make_float2(tau_yz, tau_zz);
    }
}