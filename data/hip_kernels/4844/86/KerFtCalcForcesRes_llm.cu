#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerFtCalcForcesRes(unsigned ftcount, bool simulate2d, double dt, const float3 *ftoomega, const float3 *ftovel, const double3 *ftocenter, const float3 *ftoforces, float3 *ftoforcesres, double3 *ftocenterres)
{
    const unsigned cf = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread only processes work within bounds
    if (cf < ftcount) {
        // Load fomega and compute with omegaace
        float3 fomega = ftoomega[cf];
        const float3 omegaace = ftoforces[cf * 2 + 1];
        fomega.x += dt * omegaace.x;
        fomega.y += dt * omegaace.y;
        fomega.z += dt * omegaace.z;

        // Load fvel and face
        float3 fvel = ftovel[cf];
        float3 face = ftoforces[cf * 2];

        // Apply 2D simulation constraints
        if (simulate2d) {
            face.y = 0;
            fomega.x = 0;
            fomega.z = 0;
            fvel.y = 0;
        }

        // Update fcenter using fvel
        double3 fcenter = ftocenter[cf];
        fcenter.x += dt * fvel.x;
        fcenter.y += dt * fvel.y;
        fcenter.z += dt * fvel.z;

        // Update fvel with face
        fvel.x += dt * face.x;
        fvel.y += dt * face.y;
        fvel.z += dt * face.z;

        // Output results
        ftoforcesres[cf * 2] = fomega;
        ftoforcesres[cf * 2 + 1] = fvel;
        ftocenterres[cf] = fcenter;
    }
}