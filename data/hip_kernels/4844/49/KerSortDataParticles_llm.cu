#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerSortDataParticles(unsigned n, unsigned pini, const unsigned *sortpart, const float3 *a, float3 *a2) 
{
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (p < n) {
        unsigned oldpos = (p < pini ? p : sortpart[p]);
        a2[p] = a[oldpos];
    }
}