#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerInOutInterpolateTime(unsigned npt, double fxtime, const float *vel0, const float *vel1, float *vel)
{
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure thread processes only valid particles
    if (p < npt) {
        float v0 = vel0[p];
        float v1 = vel1[p];
        // Interpolate velocity
        vel[p] = fxtime * (v1 - v0) + v0;
    }
}