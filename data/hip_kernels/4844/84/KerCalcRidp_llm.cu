#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerCalcRidp(unsigned n, unsigned ini, unsigned idini, unsigned idfin, const unsigned *idp, unsigned *ridp)
{
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index
    if (p < n) {
        p += ini; // Adjust particle index with ini
        const unsigned id = idp[p]; // Fetch id for particle
        if (idini <= id && id < idfin) {
            unsigned ridx = id - idini; // Precompute output index
            ridp[ridx] = p; // Directly assign to ridp
        }
    }
}