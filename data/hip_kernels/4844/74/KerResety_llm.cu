#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerResety(unsigned n, unsigned ini, float3 *v)
{
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global particle index

    // Check boundary condition before assignment
    if (p < n) {
        v[p + ini].y = 0; // Reset y component
    }
}