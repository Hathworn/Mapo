#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerCalcRidp(unsigned n, unsigned ini, unsigned idini, unsigned idfin, const unsigned *idp, unsigned *ridp)
{
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global particle index.
    if (p < n)
    {
        p += ini; // Adjust index by initial offset.
        unsigned id = idp[p]; // Fetch particle id.
        if (idini <= id && id < idfin)
        {
            atomicAdd(&ridp[id - idini], p); // Use atomic operation to prevent race conditions.
        }
    }
}