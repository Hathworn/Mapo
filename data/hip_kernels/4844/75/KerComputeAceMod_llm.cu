#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerComputeAceMod(unsigned n, const float3 *ace, float *acemod)
{
    // Calculate particle index based on unique thread identifier
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if particle index is within bounds
    if (p < n) {
        // Load ace vector into registers
        const float3 r = ace[p];
        
        // Calculate and store the squared magnitude of the ace vector
        acemod[p] = r.x * r.x + r.y * r.y + r.z * r.z;
    }
}