#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerComputeVelMod(unsigned n, const float4 *vel, float *velmod)
{
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x; // -Number of particle.
    if (p < n) {
        const float4 r = vel[p];
        // Compute velocity magnitude using inline computation
        velmod[p] = fmaf(r.x, r.x, fmaf(r.y, r.y, r.z * r.z));
    }
}