#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KerSortDataParticles(unsigned n, unsigned pini, const unsigned *sortpart, const double2 *a, const double *b, const float4 *c, double2 *a2, double *b2, float4 *c2)
{
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < n) {
        unsigned oldpos = (p < pini ? p : sortpart[p]);
        // Cache read from global memory into registers
        double2 a_val = a[oldpos];
        double b_val = b[oldpos];
        float4 c_val = c[oldpos];

        // Write back to global memory
        a2[p] = a_val;
        b2[p] = b_val;
        c2[p] = c_val;
    }
}