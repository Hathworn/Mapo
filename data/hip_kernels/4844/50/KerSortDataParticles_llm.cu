#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerSortDataParticles(unsigned n, unsigned pini, const unsigned *sortpart, const float *a, float *a2) {
    const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; // Particle number
    if (p < n) {
        unsigned oldpos = sortpart[p]; // Fetching values without the condition
        if (p < pini) oldpos = p; // Inline replacement for conditional assignment
        a2[p] = a[oldpos];
    }
}