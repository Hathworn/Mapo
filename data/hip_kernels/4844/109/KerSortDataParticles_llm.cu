#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerSortDataParticles(unsigned n, unsigned pini, const unsigned *sortpart, const float3 *a, float3 *a2)
{
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread is within bounds
    if (p < n) {
        unsigned oldpos = (p < pini) ? p : sortpart[p];

        // Perform the assignment with minimal indexing
        a2[p] = a[oldpos];
    }
}