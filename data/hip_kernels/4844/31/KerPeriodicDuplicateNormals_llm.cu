#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerPeriodicDuplicateNormals(unsigned n, unsigned pini, const unsigned *listp, float3 *normals, float3 *motionvel) {
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x; // Compute particle index.
    if (p < n) {
        unsigned pnew = p + pini;
        unsigned rp = listp[p];
        unsigned pcopy = rp & 0x7FFFFFFF;
        normals[pnew] = normals[pcopy]; // Duplicate normals.
        if (motionvel) {
            motionvel[pnew] = motionvel[pcopy]; // Duplicate motion velocities if not null.
        }
    }
}