#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KerSortDataParticles(unsigned n, unsigned pini, const unsigned *sortpart, const float *a, const float *b, float *a2, float *b2)
{
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x; // Calculate particle number
    if (p < n) {
        unsigned oldpos = (p < pini) ? p : sortpart[p]; // Determine old position
        a2[p] = a[oldpos];
        b2[p] = b[oldpos];
    }
}