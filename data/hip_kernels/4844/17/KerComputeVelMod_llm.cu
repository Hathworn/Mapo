#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerComputeVelMod(unsigned n, const float4 *vel, float *velmod)
{
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x; // Calculate particle index
    if(p < n)
    {
        const float4 r = vel[p];
        // using intrinsic function to compute dot product for efficiency
        velmod[p] = __dot3f(r, r); 
    }
}