#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerSortDataParticles(unsigned n, unsigned pini, const unsigned *sortpart, const float *a, const float *b, float *a2, float *b2)
{
    const unsigned p = blockIdx.x * blockDim.x + threadIdx.x; // Compute particle index
    if (p < n) {
        const unsigned oldpos = (p < pini ? p : sortpart[p]); // Determine original position of the particle
        a2[p] = a[oldpos];
        b2[p] = b[oldpos];
    }
}