#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerInOutUpdateVelrhopM1(unsigned n, const int *inoutpart, const float4 *velrhop, float4 *velrhopm1)
{
    // Calculate the global thread index
    const unsigned cp = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if within bounds
    if (cp < n) {
        const unsigned p = inoutpart[cp];
        // Update velrhopm1 with values from velrhop
        velrhopm1[p] = velrhop[p];
    }
}