#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel launch using dynamic block size adjustment
__global__ void KerResety(unsigned n, unsigned ini, float3 *v) {
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x;
    if (p < n) {
        v[p + ini].y = 0;
    }
}
```
