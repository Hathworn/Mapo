#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KerResety(unsigned n, unsigned ini, float3 *v)
{
    unsigned p = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index
    if (p < n) // Check if thread index is within bounds
    {
        v[p + ini].y = 0; // Reset y component
    }
}