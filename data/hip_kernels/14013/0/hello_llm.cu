#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 3

__global__ void hello()
{
    // Use variables to avoid repeated access to blockIdx and threadIdx
    int blockIndex = blockIdx.x;
    int threadIndex = threadIdx.x;

    // Use conditional execution to minimize unnecessary printf calls
    if (threadIndex < BLOCK_WIDTH && blockIndex < NUM_BLOCKS) {
        printf("Hello world! I'm the %dth thread in %dth block. \n", threadIndex, blockIndex);
    }
}