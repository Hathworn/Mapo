#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateVelocity_k(float2 *v, float *vx, float *vy, int dx, int pdx, int dy, int lb, size_t pitch) {
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidyStart = blockIdx.y * (lb * blockDim.y);
    float scale = 1.f / (dx * dy); // Precompute scale factor

    // Check if thread is within the domain in X
    if (gtidx < dx) {
        for (int p = 0; p < lb; p++) {
            int fi = gtidyStart + threadIdx.y * lb + p; // Calculate global index in y

            // Check if thread is within the domain in Y
            if (fi < dy) {
                int fjr = fi * pdx + gtidx;
                
                // Load vx and vy terms
                float vxterm = vx[fjr];
                float vyterm = vy[fjr];

                // Normalize and store
                float2 nvterm;
                nvterm.x = vxterm * scale;
                nvterm.y = vyterm * scale;

                // Update velocity
                float2 *fj = (float2*)((char*)v + fi * pitch) + gtidx;
                *fj = nvterm;
            }
        }
    }
}