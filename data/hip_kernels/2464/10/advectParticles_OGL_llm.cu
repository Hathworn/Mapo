#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void advectParticles_OGL(float2 *part, float2 *v, int dx, int dy, float dt, int lb, size_t pitch) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    
    // Check if within domain bounds in X
    if (gtidx >= dx) return;
    
    float2 pterm, vterm;

    for (int p = 0; p < lb; p++) {
        int fi = gtidy + p;
        
        // Check if within domain bounds in Y
        if (fi >= dy) continue;
        
        int fj = fi * dx + gtidx;
        pterm = part[fj];
        
        int xvi = static_cast<int>(pterm.x * dx);
        int yvi = static_cast<int>(pterm.y * dy);
        vterm = *((float2*)((char*)v + yvi * pitch) + xvi);

        // Update particle position and ensure it wraps correctly
        pterm.x = fmodf(pterm.x + dt * vterm.x + 1.f, 1.f);
        pterm.y = fmodf(pterm.y + dt * vterm.y + 1.f, 1.f);

        part[fj] = pterm;
    }
}