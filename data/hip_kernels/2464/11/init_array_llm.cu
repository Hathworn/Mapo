#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use register for temporary storage to optimize access time
    int local_factor = *factor;
    
    // Unroll loops for improved performance
    for (int i = 0; i < num_iterations; i += 4)
    {
        g_data[idx] += local_factor;
        g_data[idx] += local_factor;
        g_data[idx] += local_factor;
        g_data[idx] += local_factor;
    }
}