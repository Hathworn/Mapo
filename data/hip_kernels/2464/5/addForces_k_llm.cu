#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addForces_k(float2 *v, int dx, int dy, int spx, int spy, float fx, float fy, int r, size_t pitch) {
    // Calculate global thread coordinates
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure thread is within bounds
    if (tx >= dx || ty >= dy) return;

    // Calculate the position in memory
    float2 *fj = (float2*)((char*)v + (ty + spy) * pitch) + tx + spx;

    // Read the current value
    float2 vterm = *fj;

    // Apply force computation
    tx -= r; 
    ty -= r;
    float s = 1.f / (1.f + tx*tx*tx*tx + ty*ty*ty*ty);
    vterm.x += s * fx;
    vterm.y += s * fy;

    // Write the result back to memory
    *fj = vterm;
}