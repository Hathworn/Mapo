#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "diffuseProject_k_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float2 *vx = NULL;
hipMalloc(&vx, XSIZE*YSIZE);
float2 *vy = NULL;
hipMalloc(&vy, XSIZE*YSIZE);
int dx = 1;
int dy = 1;
float dt = 1;
float visc = 1;
int lb = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
diffuseProject_k<<<gridBlock, threadBlock>>>(vx,vy,dx,dy,dt,visc,lb);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
diffuseProject_k<<<gridBlock, threadBlock>>>(vx,vy,dx,dy,dt,visc,lb);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
diffuseProject_k<<<gridBlock, threadBlock>>>(vx,vy,dx,dy,dt,visc,lb);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}