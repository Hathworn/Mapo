#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAdd(const float* A, const float* B, float* C, int N)
{
    // Calculate the global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Use warp-level optimization by iterating over array with stride
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        C[idx] = A[idx] + B[idx];
    }
}