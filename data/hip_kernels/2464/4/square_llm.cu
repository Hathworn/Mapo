#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square(int *array, int arrayCount)
{
    // Calculate global thread ID
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Check if the global thread ID is within bounds
    if (idx < arrayCount) {
        int value = array[idx];
        array[idx] = value * value; // Perform squaring operation
    }
}