#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Initial write
    *ptr = pattern;

    while(ptr < end_ptr)
    {
        // Calculate new pointer position using mask
        unsigned int* new_ptr = (unsigned int*)((unsigned long)ptr | mask);

        // Ensure progression to avoid infinite loop and check bounds
        if(new_ptr >= end_ptr)
        {
            break;
        }

        // Write pattern at new pointer position
        *new_ptr = pattern;

        // Update ptr and pattern for next iteration
        ptr = new_ptr;
        pattern <<= 1;
        mask <<= 1;
    }
}