#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Early exit if the current block's start pointer exceeds the end pointer
    if (ptr >= (unsigned long*) end_ptr) {
        return;
    }

    // Iterate over each element in the block and write its address for the whole block
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned long); i += blockDim.x) {
        ptr[i] = (unsigned long) &ptr[i];
    }
}