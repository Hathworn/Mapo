#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr) {
    // Pre-calculate shared values
    unsigned int* orig_ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* ptr = orig_ptr;
    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);
    unsigned int pattern = 1;
    unsigned long mask = 4;

    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    *ptr = pattern;

    while (ptr < block_end) {
        // Optimize pointer calculation
        unsigned long new_ptr_value = ((unsigned long)orig_ptr) | mask;
        ptr = (unsigned int*) new_ptr_value;
        if (ptr == orig_ptr) {
            mask <<= 1; // Efficiently shift mask
            continue;
        }
        if (ptr >= block_end) {
            break;
        }

        *ptr = pattern;

        // Efficiently shift pattern and mask
        pattern <<= 1;
        mask <<= 1;
    }
    return;
}