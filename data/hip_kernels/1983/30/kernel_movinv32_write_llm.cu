#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Global kernel function optimized for coalesced memory access
__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset) {
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Ensure we do not exceed the boundary
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int k = offset;
    unsigned int pat = pattern;
    unsigned int stride = BLOCKSIZE / sizeof(unsigned int);
    
    // Reduce loop iteration frequency for performance
    #pragma unroll
    for (unsigned int i = 0; i < stride; i++){
        ptr[i] = pat;
        k++;
        if (k >= 32) {
            k = 0;
            pat = lb;
        } else {
            pat = (pat << 1) | sval;
        }
    }
}