#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    // Calculate global thread index for coalesced access
    unsigned int idx = blockIdx.x * BLOCKSIZE + threadIdx.x * sizeof(unsigned int);

    // Convert _ptr to unsigned int pointer and calculate the target address
    unsigned int* ptr = (unsigned int*)(_ptr + idx);
    
    // Ensure thread accesses within bounds
    if ((char*)ptr >= end_ptr) {
        return;
    }
    
    // Write pattern to memory in a coalesced and efficient way using threads
    ptr[0] = pattern;
}