#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    unsigned int global_idx = blockIdx.x * BLOCKSIZE;

    // Convert char pointers to unsigned int pointers for indexing
    unsigned int* ptr = (unsigned int*)(_ptr + global_idx);
    unsigned int* start_ptr = (unsigned int*)_start_ptr;

    // Ensure the pointer is within bounds
    if (ptr + BLOCKSIZE / sizeof(unsigned int) > (unsigned int*)end_ptr) {
        return;
    }

    // Loop unrolling for performance
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        ptr[i] = start_ptr[i];
    }

    return;
}