#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    // Check boundary condition at the start
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int i = offset;
    // Unroll the loop to reduce number of iterations
    #pragma unroll
    for (; i < BLOCKSIZE/sizeof(unsigned int); i += MOD_SZ) {
        ptr[i] = p1;
    }

    // Separate loop to handle remaining operations
    #pragma unroll
    for (i = 0; i < BLOCKSIZE/sizeof(unsigned int); i++) {
        if (i % MOD_SZ != offset) {
            ptr[i] = p2;
        }
    }
    return;
}