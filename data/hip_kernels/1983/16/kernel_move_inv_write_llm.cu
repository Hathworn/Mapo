#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    // Calculate global thread index
    unsigned int globalIdx = blockIdx.x * BLOCKSIZE + threadIdx.x * sizeof(unsigned int);
    unsigned int* ptr = (unsigned int*) (_ptr + globalIdx);

    // Check boundary condition
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Write pattern in parallel using threads
    ptr[0] = pattern;
}