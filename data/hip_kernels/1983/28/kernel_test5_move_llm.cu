#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check boundary condition for the pointer
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Calculate half count for block data
    unsigned int half_count = BLOCKSIZE / sizeof(unsigned int) / 2;
    unsigned int* ptr_mid = ptr + half_count;

    // Use a single loop to minimize overhead and improve efficiency
    #pragma unroll
    for (unsigned int i = 0; i < half_count; i++) {
        // Simultaneously handle both mid and initial sections
        if (i < half_count - 8) {
            ptr[i + 8] = ptr_mid[i];  // First half copying 
        }
        if (i < 8) {
            ptr[i] = ptr_mid[half_count - 8 + i];  // Last 8 elements handling
        }
        ptr_mid[i] = ptr[i];  // Move first half to second
    }
    
    return;
}