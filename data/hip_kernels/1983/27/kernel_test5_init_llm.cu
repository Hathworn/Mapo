#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test5_init(char* _ptr, char* end_ptr)
{
    // Calculate the starting index for this block
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check if the pointer is beyond the end pointer
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }
    
    unsigned int p1 = 1;

    // Loop unrolling for improved performance
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i += 16) {
        unsigned int p2 = ~p1;

        // Store data in chunks
        ptr[i] = p1; ptr[i+1] = p1; ptr[i+2] = p2; ptr[i+3] = p2;
        ptr[i+4] = p1; ptr[i+5] = p1; ptr[i+6] = p2; ptr[i+7] = p2;
        ptr[i+8] = p1; ptr[i+9] = p1; ptr[i+10] = p2; ptr[i+11] = p2;
        ptr[i+12] = p1; ptr[i+13] = p1; ptr[i+14] = p2; ptr[i+15] = p2;

        // Update p1 and reset if it becomes zero
        p1 = (p1 << 1) | (p1 == 0);
    }
}