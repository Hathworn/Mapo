#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* __restrict__ original, int64_t* __restrict__ to_normalize, int64_t batch_index, int64_t class_index) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Unrolled loop for improved memory access pattern and coalescing
    for (; idx < num_elements; idx += stride) {
        int base = idx * 3;
        to_normalize[base] = batch_index;
        to_normalize[base + 1] = class_index;
        to_normalize[base + 2] = static_cast<int64_t>(original[idx]);
    }
}