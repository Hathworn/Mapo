#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel using registers for temporary storage
__device__ int2 devInt2[10];
__global__ void regOperation() {
    int2 f = devInt2[1];
    devInt2[0] = f; // Use register to store and transfer data efficiently
}

__global__ void regOperation2() {
    int2 f = devInt2[1];
    devInt2[0] = f; // Optimize by handling the entire struct to reduce split accesses
}