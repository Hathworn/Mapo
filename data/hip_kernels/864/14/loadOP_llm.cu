#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Unroll loop to improve performance; use __restrict__ qualifiers to help optimizer
__global__ void loadOP() {
    // Assuming input and output are declared and accessible
    // Use registers to store intermediate results
    const float * __restrict__ input;
    float * __restrict__ output;
    
    #pragma unroll
    for (int i = 0; i < 9; ++i) {
        output[i] = Load<DF + i>(input + i);
    }
}