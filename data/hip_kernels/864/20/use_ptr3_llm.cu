#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void use_ptr3() {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Compute global index
    if (idx < 100) { // Ensure within bounds
        const_ptr[idx] = idx; // Assign index value
    }
}