#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global index and ensure within bounds
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        c[i] = a[i] + b[i];
    }
}