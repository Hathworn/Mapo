#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define RED 2
#define GREEN 1
#define BLUE 0

using namespace std;

extern "C"
__global__ void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width) {
        // Calculate linear index once to improve performance
        int index = row * width + col;
        int rgbIndex = index * 3;

        // Optimize by avoiding repeated calculation
        unsigned char r = imageInput[rgbIndex + RED];
        unsigned char g = imageInput[rgbIndex + GREEN];
        unsigned char b = imageInput[rgbIndex + BLUE];

        // Compute grayscale value
        imageOutput[index] = r * 0.299f + g * 0.587f + b * 0.114f;
    }
}