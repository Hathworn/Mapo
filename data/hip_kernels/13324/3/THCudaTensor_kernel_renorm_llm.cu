#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_renorm(float *data, const float value, const long size, const float maxnorm)
{
    extern __shared__ float buffer[]; // Use dynamic shared memory for better resource utilization
    long tx = threadIdx.x;
    long bx = blockIdx.x;
    long step = blockDim.x;
    float *row = data + size * bx;

    // Initialize buffer
    buffer[tx] = 0.0f;

    // Compute the norm of axis
    for (long i = tx; i < size; i += step)
    {
        buffer[tx] += powf(fabsf(row[i]), value);
    }

    // Reduction using shared memory
    __syncthreads();
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tx < stride)
        {
            buffer[tx] += buffer[tx + stride];
        }
        __syncthreads();
    }

    // Normalization and renorm if necessary
    if (tx == 0)
    {
        float norm = powf(buffer[0], 1.0f / value);
        buffer[0] = (norm > maxnorm) ? (maxnorm / (norm + 1e-7f)) : 1.0f;
    }

    // Synchronize to ensure buffer[0] is updated
    __syncthreads();

    // Renormalize if required
    float norm = buffer[0];
    if (norm < 1.0f)
    {
        for (long i = tx; i < size; i += step)
        {
            row[i] *= norm;
        }
    }
}