#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv2genericrev(float *input, float *kernel, float *output, int input_n, int input_h, int input_w, int kernel_n, int kernel_h, int kernel_w, float alpha, int stride_h, int stride_w) 
{
    // compute output dimensions
    int output_h = input_h - (kernel_h - 1) * stride_h;
    int output_w = input_w - (kernel_w - 1) * stride_w;

    // process one output defined by block Ids
    int kk = blockIdx.x;  // kernel index
    int ii = blockIdx.y;  // input index

    // batch id
    int batch = threadIdx.z;

    // kernel id
    int kid = threadIdx.x;
    int nkids = blockDim.x;

    // thread ID
    int tid = kid + batch*blockDim.x;
    int nthreads = blockDim.x * blockDim.z;

    // pointer to this thread's output position
    output = output + (kk * input_n + ii) * output_h * output_w;

    // allocate shared memory for outputs
    extern __shared__ float shared_output[];

    // generate thread outputs in shared memory
    float *output_s = shared_output + tid * output_w * output_h;

    // convolution loop
    int xx, kx, ky;
    float output_p[output_w]; // local output accumulator

    // iterate over the input height dimension
    for (int yy = threadIdx.y; yy < output_h; yy += blockDim.y) {
        // load the current output pointer
        float *input_p = input + ((ii + batch * input_n) * input_h + yy * stride_h) * input_w;
        float *kernel_p = kernel + ((kk + batch * kernel_n) * kernel_h) * kernel_w;

        // reset the output accumulator
        for (xx = 0; xx < output_w; xx++) {
            output_p[xx] = 0.0f;
        }

        // perform the convolution
        for (ky = 0; ky < kernel_h; ky++) {
            for (kx = kid; kx < kernel_w; kx += nkids) {
                float val = kernel_p[ky * kernel_w + kx];
                for (xx = 0; xx < output_w; xx++) {
                    output_p[xx] += input_p[xx * stride_w + kx] * val;
                }
            }
            input_p += input_w; // move to next row in input
        }

        // store the output in shared memory
        for (xx = 0; xx < output_w; xx++) {
            output_s[yy * output_w + xx] = output_p[xx];
        }
    }

    __syncthreads();

    // reduce and write back
    if (threadIdx.y == 0 && tid < output_h * output_w) {
        // reduce outputs across all threads
        for (int k = nthreads; k < nthreads; k += nthreads) {
            shared_output[tid] += shared_output[k * output_h * output_w + tid];
        }
        __syncthreads();

        // add existing output and write back via alpha scaling
        output[tid] += alpha * shared_output[tid];
    }
}