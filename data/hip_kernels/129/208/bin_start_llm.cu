#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bin_start(int *binStart, int *binEnd, int *partBin, int nparts)
{
    extern __shared__ int sharedBin[];    
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int bin;

    if (index < nparts) {
        bin = partBin[index];
        sharedBin[threadIdx.x + 1] = bin;

        // First thread loads the previous bin if necessary
        if (threadIdx.x == 0) {
            sharedBin[0] = (index > 0) ? partBin[index - 1] : -1;
        }
    }
    __syncthreads();

    if (index < nparts) {
        bin = partBin[index];

        // Check if the current particle is the start of a new bin
        if (index == 0 || bin != sharedBin[threadIdx.x]) {
            binStart[bin] = index;
            if (index > 0) {
                binEnd[sharedBin[threadIdx.x]] = index;
            }
        }

        // Handle the last particle separately
        if (index == nparts - 1) {
            binEnd[bin] = nparts;
        }
    }
}
```
