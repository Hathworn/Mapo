#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(int *vector, int N, int val)
{
    // Use `int` to `register` for faster access and reduce global memory transactions
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use branchless programming to avoid if-condition
    int assign_value = (idx < N) ? val : 0; // Default `0` will not matter if idx >= N
    vector[idx] = assign_value;

    // Alternatively, if `vector` size is guaranteed by the caller, remove the conditional entirely
    // vector[idx] = val;
}