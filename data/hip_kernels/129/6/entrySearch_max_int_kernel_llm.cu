#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void entrySearch_max_int_kernel(int *g_iarr, int *g_maxarr, int size) {
    // Create shared memory
    extern __shared__ int sarr_int[];

    // Load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // Use a single load instead of conditional statements
    sarr_int[tid] = (i < size) ? g_iarr[i] : INT_MIN;
    if (i + blockDim.x < size) {
        sarr_int[tid] = max(sarr_int[tid], g_iarr[i + blockDim.x]);
    }

    __syncthreads();

    // Do comparison in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sarr_int[tid] = max(sarr_int[tid], sarr_int[tid + s]);
        }
        __syncthreads();
    }

    // Write result for this block to global mem
    if (tid == 0) {
        g_maxarr[blockIdx.x] = sarr_int[0];
    }
}