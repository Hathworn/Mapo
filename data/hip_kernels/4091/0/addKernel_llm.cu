#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global thread index for processing larger arrays
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) { // Check bounds to prevent out-of-bounds access
        c[i] = a[i] + b[i];
    }
}