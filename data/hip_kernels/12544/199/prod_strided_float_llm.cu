#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prod_strided_float(int n, int xOffset, float *dx, int incx, float *result) {
    // Shared memory reduction
    __shared__ float partialProd[256];
    int tid = threadIdx.x;
    partialProd[tid] = 1.0f;

    // Compute product for each thread's segment
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        if (i >= xOffset && i % incx == 0) {
            partialProd[tid] *= dx[i];
        }
    }
    __syncthreads();

    // Warp-level reduction
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        if (tid < offset) {
            partialProd[tid] *= partialProd[tid + offset];
        }
        __syncthreads();
    }

    // Write the result from the first thread of each block to global memory
    if (tid == 0) {
        atomicMul(result, partialProd[0]);
    }
}