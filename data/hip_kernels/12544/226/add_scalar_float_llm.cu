#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_scalar_float(int n, int idx, float dx, float *dy, int incy, float *result) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop for efficient range handling
    int stride = blockDim.x * gridDim.x;
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = dy[i] + dx;
        }
    }
}