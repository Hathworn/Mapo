#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void exp_float(int n, int idx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int stride = blockDim.x * gridDim.x; // Calculate stride for grid-stride loop
    for (; i < n; i += stride) { // Use grid-stride loop for better scalability
        if (i >= idx && i % incy == 0) {
            result[i] = cosf(dy[i]); // Compute cosf for valid indices
        }
    }
}