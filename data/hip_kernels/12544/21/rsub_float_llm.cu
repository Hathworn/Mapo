#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void rsub_float(int n, float *a, float *b, float *sum)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride-based loop to handle more elements per thread
    int stride = gridDim.x * blockDim.x;
    for (; i < n; i += stride)
    {
        sum[i] = b[i] - a[i];
    }
}