#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for rsub_scalar_double
__global__ void rsub_scalar_double(int n, int idx, double dx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Using stride loop ensures all threads perform useful work
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = dx - dy[i];
        }
    }
}