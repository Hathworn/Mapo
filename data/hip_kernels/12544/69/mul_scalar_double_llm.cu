#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void mul_scalar_double(int n, int idx, double dx, double *dy, int incy, double *result) {
    // Optimize loop by using stride to avoid redundant calculations
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0)
            result[i] = dy[i] * dx;
    }
}