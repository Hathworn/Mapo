#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void sign_double(int n,int idx,double *dy,int incy,double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Unrolled loop to enhance performance
    for (; i < n; i += stride) {
        if(i >= idx && i % incy == 0) {
            double x = dy[i];
            result[i] = (x > 0) - (x < 0);
        }
    }
}