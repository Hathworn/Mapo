#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void neg_float(int n, int idx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int stride = blockDim.x * gridDim.x; // Total number of threads

    while (i < n) { // Iterate over elements with stride
        if (i >= idx && i % incy == 0) {
            result[i] = -dy[i]; // Assign negated value
        }
        i += stride; // Move to the next element handled by this thread
    }
}