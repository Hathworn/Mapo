#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize the kernel function
__global__ void eq_strided_float(int n, int xOffset, int yOffset, float *dx, float *dy, int incx, int incy, float *result) {
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the global index is within bounds and strides are met
    if(i < n && i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
        result[i] = dy[i] == dx[i];
    }
}