#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void mul_strided_float(int n, int xOffset, int yOffset, float *dx, float *dy, int incx, int incy, float *result) {
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride-based loop
    int stride = blockDim.x * gridDim.x;

    // Iterate over the elements
    for (; i < n; i += stride) {
        // Ensure indices and strides are respected
        if (i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dx[i] * dy[i];
        }
    }
}