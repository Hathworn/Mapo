#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void rdiv_strided_double(int n, int xOffset, int yOffset, double *dx, double *dy, int incx, int incy, double *result) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop with stride for handling larger data
    while (i < n) {
        // Check strided conditions
        if (i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dx[i] - dy[i];
        }
        // Increment index with grid stride
        i += blockDim.x * gridDim.x;
    }
}