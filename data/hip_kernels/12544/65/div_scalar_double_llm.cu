#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void div_scalar_double(int n, int idx, double dx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Compute global thread index
    if(i < n && i >= idx && i % incy == 0) {        // Check bounds and conditions
        result[i] = dy[i] / dx;                     // Perform division
    }
    // Use stride looping for larger datasets
    for (i += blockDim.x * gridDim.x; i < n; i += blockDim.x * gridDim.x) {
        if(i >= idx && i % incy == 0)
            result[i] = dy[i] / dx;
    }
}