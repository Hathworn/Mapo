#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_float(int n, float *a, float *b, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride to process larger data sets
    for(int idx = i; idx < n; idx += stride) { // Loop over elements in steps of stride
        sum[idx] = a[idx] * b[idx];
    }
}