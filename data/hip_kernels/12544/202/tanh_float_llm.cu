#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tanh_float(int n, int idx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop unroll to improve memory access coalescing
    while (i < n) {
        if (i >= idx && i % incy == 0)
            result[i] = tanh(dy[i]);
        i += stride;
    }
}