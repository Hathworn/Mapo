#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized Kernel Function with Improved Memory Access Pattern
__global__ void sqrt_float(int n, int idx, float *dy, int incy, float *result) {
    // Use shared memory when applicable
    extern __shared__ float shared_dy[];

    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

    if (thread_id < n) {
        // Only load relevant data into shared memory
        if (thread_id >= idx && thread_id % incy == 0) {
            shared_dy[threadIdx.x] = dy[thread_id];
            __syncthreads();

            // Perform computation using shared memory
            result[thread_id] = sqrtf(shared_dy[threadIdx.x]);
        }
    }
}