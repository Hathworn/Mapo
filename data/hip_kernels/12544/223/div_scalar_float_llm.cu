#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void div_scalar_float(int n, int idx, float dx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop using strided access across threads
    for (; i < n; i += blockDim.x * gridDim.x) {
        if (i >= idx && i % incy == 0) {
            result[i] = dy[i] / dx;
        }
    }
}