#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void tanh_float(int n, int idx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Iterate over the thread-stride loop
    while (i < n) {
        if (i >= idx && i % incy == 0) 
            result[i] = tanh(dy[i]);
        
        i += blockDim.x * gridDim.x;
    }
}