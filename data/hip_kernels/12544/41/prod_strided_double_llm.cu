#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void prod_strided_double(int n, int xOffset, double *dx, int incx, double result) {
    // Local result for each thread to avoid collisions
    double local_result = 1.0;
    
    // Use a grid stride loop for better performance
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        if(i >= xOffset && (i - xOffset) % incx == 0)  // Corrected stride condition
            local_result *= dx[i];
    }

    // Atomic operation to reduce results from all threads
    atomicMul(&result, local_result);
}

__device__ void atomicMul(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*) address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val * __longlong_as_double(assumed)));
    } while (assumed != old);
}