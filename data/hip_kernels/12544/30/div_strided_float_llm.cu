#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void div_strided_float(int n, int xOffset, int yOffset, float *dx, float *dy, int incx, int incy, float *result) {
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use strided loop with the total threads count as the stride
    int stride = blockDim.x * gridDim.x;
    while (i < n) {
        // Check conditions and perform division if met
        if(i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dy[i] / dx[i];
        }
        i += stride; // Move to the next element in the stride
    }
}