#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void eq_strided_float(int n, int xOffset, int yOffset, float *dx, float *dy, int incx, int incy, float *result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Utilize shared memory to reduce global memory access
    __shared__ float shared_dx[256];
    __shared__ float shared_dy[256];

    for (int i = idx; i < n; i += blockDim.x * gridDim.x) {
        if (i >= max(xOffset, yOffset) && i % incx == 0 && i % incy == 0) {
            // Load data into shared memory
            shared_dx[threadIdx.x] = dx[i];
            shared_dy[threadIdx.x] = dy[i];
            __syncthreads();

            // Perform computation
            result[i] = shared_dy[threadIdx.x] == shared_dx[threadIdx.x];
        }
    }
}