#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void exp_float(int n, int idx, float *dy, int incy, float *result) {
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check bounds and condition
    if (i < n && i >= idx && i % incy == 0) {
        // Perform the exponentiation
        result[i] = expf(dy[i]);
    }
}