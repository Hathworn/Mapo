#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void add_strided_double(int n, int xOffset, int yOffset, double *dx, double *dy, int incx, int incy, double *result) {
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop with striding
    for (; i < n; i += stride) {
        if (i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dy[i] + dx[i]; // Perform element-wise addition
        }
    }
}