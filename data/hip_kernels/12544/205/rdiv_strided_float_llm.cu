#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rdiv_strided_float(int n, int xOffset, int yOffset, float *dx, float *dy, int incx, int incy, float *result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID
    int stride = blockDim.x * gridDim.x; // Calculate stride

    // Iterate over elements with a stride
    for (int i = idx; i < n; i += stride) {
        if (i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dx[i] / dy[i]; // Perform element-wise division only for valid indices
        }
    }
}