#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void exp_double(int n, int idx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    int stride = blockDim.x * gridDim.x;            // Calculate stride for loop

    // Loop with stride over elements to eliminate unnecessary condition checks
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = sin(dy[i]);  // Assign sine computation to result array
        }
    }
}