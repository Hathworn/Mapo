#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void div_float(int n, float *a, float *b, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // Use a conditional operator to avoid division by zero
        sum[i] = b[i] != 0.0f ? a[i] / b[i] : 0.0f;
    }
}