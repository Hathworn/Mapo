#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sign_double(int n, int idx, double *dy, int incy, double *result) {
    // Use the shared memory for faster data access
    extern __shared__ double shared_dy[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    // Load data into shared memory
    if (i < n) {
        shared_dy[tid] = dy[i];
    }
    __syncthreads();

    // Perform sign calculation using shared memory
    while (i < n) {
        if (i >= idx && i % incy == 0) {
            double x = shared_dy[tid];
            result[i] = (x > 0) - (x < 0);
        }
        i += blockDim.x * gridDim.x;
    }
}