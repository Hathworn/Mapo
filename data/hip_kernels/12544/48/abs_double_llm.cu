#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void abs_double(int n, int idx, double *dy, int incy, double *result) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Iterate over the array using stride
    while (i < n) {
        // Check conditions and compute
        if (i >= idx && i % incy == 0) {
            result[i] = abs(dy[i]);
        }
        // Move to the next element by stride
        i += blockDim.x * gridDim.x;
    }
}