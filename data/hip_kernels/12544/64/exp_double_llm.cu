#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void exp_double(int n, int idx, double *dy, int incy, double *result) {
    // Calculate the unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Stride loop for better access pattern
    int stride = blockDim.x * gridDim.x;
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = exp(dy[i]);
        }
    }
}