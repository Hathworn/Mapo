#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void mul_double(int n, double *a, double *b, double *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Loop over elements with a stride to utilize all available threads
    for (; i < n; i += stride)
    {
        sum[i] = a[i] * b[i];
    }
}