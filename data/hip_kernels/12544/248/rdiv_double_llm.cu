#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rdiv_double(int n, double *a, double *b, double *sum)
{
    // Calculate global index for this thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use continuous threads to reduce overall loop iterations for larger problem sizes
    int stride = gridDim.x * blockDim.x;

    // Iterate over the array with the thread stride for better throughput
    for (int idx = i; idx < n; idx += stride) {
        sum[idx] = b[idx] / a[idx];
    }
}