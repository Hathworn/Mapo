#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

// Optimized kernel function for float multiplication
__global__ void mul_float(int n, float *a, float *b, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // Using registers for better performance
        float ai = a[i];
        float bi = b[i];
        sum[i] = ai * bi;
    }
}