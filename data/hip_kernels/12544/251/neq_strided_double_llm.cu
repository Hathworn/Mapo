#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void neq_strided_double(int n, int xOffset, int yOffset, double *dx, double *dy, int incx, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single loop to reduce thread divergence and improve readability
    while (i < n) {
        if (i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dy[i] != dx[i];
        }
        i += blockDim.x * gridDim.x; // Move to the next index handled by this thread
    }
}