#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void sign_float(int n, int idx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop through array with stride to use coalesced reads
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            float x = dy[i];
            result[i] = (x > 0) - (x < 0); // Assign sign of x
        }
    }
}