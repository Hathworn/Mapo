#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void exp_double(int n, int idx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use stride loop to efficiently iterate over elements
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = sin(dy[i]);
        }
    }
}