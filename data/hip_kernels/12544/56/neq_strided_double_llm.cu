#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void neq_strided_double(int n, int xOffset, int yOffset, double *dx, double *dy, int incx, int incy, double *result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Unroll the loop for improved performance
    for (int i = index; i < n; i += stride) {
        if(i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dy[i] != dx[i];
        }
    }
}