#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void rsub_scalar_float(int n, int idx, float dx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Process elements in steps of stride, reducing divergence
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = dx - dy[i];
        }
    }
}