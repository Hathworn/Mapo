#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void exp_float(int n,int idx,float *dy,int incy,float *result) {
    // Calculate the global thread index once
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    // Unroll loop to maximize parallelization
    for (; i < n; i += blockDim.x * gridDim.x) { 
        if(i >= idx && i % incy == 0)
            result[i] = sinf(dy[i]);
    }
}