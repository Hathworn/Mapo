#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void rsub_double(int n, double *a, double *b, double *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        sum[i] = b[i] - a[i];
    }
    // No loop unrolling needed as each thread handles one element for efficiency
}