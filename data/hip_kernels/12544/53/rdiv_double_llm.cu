#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void rdiv_double(int n, double *a, double *b, double *sum)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop over data with stride equal to grid size
    int gridSize = blockDim.x * gridDim.x;
    while (i < n) {
        sum[i] = b[i] / a[i];
        i += gridSize;
    }
}