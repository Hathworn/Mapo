#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void abs_float(int n, int idx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    int gridStride = blockDim.x * gridDim.x;        // Calculate grid stride
    for (; i < n; i += gridStride) {                // Use grid-stride loop
        if (i >= idx && i % incy == 0)              // Conditional execution
            result[i] = fabsf(dy[i]);               // Use fabsf for float
    }
}