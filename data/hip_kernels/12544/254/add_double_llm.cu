#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_double(int n, double *a, double *b, double *sum) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use loop to handle multiple elements per thread to improve load balance
    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x) {
        sum[idx] = a[idx] + b[idx];
    }
}