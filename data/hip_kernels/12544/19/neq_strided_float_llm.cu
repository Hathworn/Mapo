#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

__global__ void neq_strided_float(int n, int xOffset, int yOffset, float *dx, float *dy, int incx, int incy, float *result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    int stride = blockDim.x * gridDim.x; // Calculate stride

    for (int i = idx; i < n; i += stride) { // Optimized loop traversal with stride
        bool inRange = (i >= xOffset && i >= yOffset); // Check offset condition
        bool aligned = (i % incx == 0 && i % incy == 0); // Check stride alignment
        if (inRange && aligned) {
            result[i] = (dy[i] != dx[i]); // Assign result
        }
    }
}