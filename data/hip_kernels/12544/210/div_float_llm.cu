#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void div_float(int n, float *a, float *b, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // Perform the division with early return if b[i] is zero, to prevent division by zero
        float divisor = b[i];
        if (divisor != 0.0f) 
        {
            sum[i] = a[i] / divisor;
        } 
        else 
        {
            sum[i] = 0.0f; // or NaN/inf based on how you wish to handle division by zero
        }
    }
}