#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

// Optimized kernel function
__global__ void rsub_strided_double(int n, int xOffset, int yOffset, double *dx, double *dy, int incx, int incy, double *result) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a stride loop to handle the full range of data
    int stride = blockDim.x * gridDim.x;
    // Process all elements with stride
    for (; i < n; i += stride) {
        if(i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0)
            result[i] = dx[i] / dy[i];
    }
}