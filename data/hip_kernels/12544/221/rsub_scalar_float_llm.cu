#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rsub_scalar_float(int n, int idx,float dx,float *dy,int incy,float *result) {
    // Use a single variable for all loop computations
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop and perform work in shared memory if possible
    for (; i < n; i += blockDim.x * gridDim.x) {
        if (i >= idx && i % incy == 0)
            result[i] = dx - dy[i];
    }
}