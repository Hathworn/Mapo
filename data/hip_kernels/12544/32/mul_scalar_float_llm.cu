#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void mul_scalar_float(int n, int idx, float dx, float *dy, int incy, float *result) {
    // Calculate the global index once
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a loop stride to skip unnecessary iterations
    int stride = blockDim.x * gridDim.x;
    
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = dy[i] * dx;
        }
    }
}