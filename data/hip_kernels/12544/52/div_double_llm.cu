#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void div_double(int n, double *a, double *b, double *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Unrolling loop for potential efficiency
    int gridSize = blockDim.x * gridDim.x;
    while (i < n) {
        sum[i] = a[i] / b[i];
        i += gridSize;
    }
}