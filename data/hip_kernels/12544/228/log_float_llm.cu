#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void log_float(int n, int idx, float *dy, int incy, float *result) {
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop for thread efficiency
    while (i < n) {
        if (i >= idx && i % incy == 0) {
            result[i] = logf(dy[i]);
        }
        i += blockDim.x * gridDim.x;
    }
}