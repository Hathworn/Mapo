#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void add_scalar_double(int n, int idx, double dx, double *dy, int incy, double *result) {
    // Calculate the global index for each thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x; // Calculate total grid size
    
    while (i < n) { // Use while loop for grid-stride looping
        if (i >= idx && i % incy == 0) {
            result[i] = dy[i] + dx; // Perform the addition if conditions are met
        }
        i += gridSize; // Increment index by grid size for the next iteration
    }
}