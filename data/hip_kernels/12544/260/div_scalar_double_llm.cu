#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void div_scalar_double(int n, int idx, double dx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop unrolling for efficiency
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = dy[i] / dx;
        }
    }
}