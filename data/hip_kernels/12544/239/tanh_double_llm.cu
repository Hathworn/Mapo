#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tanh_double(int n, int idx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    while (i < n) {
        // Process elements conditionally within index range and step
        if (i >= idx && i % incy == 0) {
            result[i] = tanh(dy[i]);
        }
        i += stride;
    }
}