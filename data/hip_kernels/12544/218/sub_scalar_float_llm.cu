#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sub_scalar_float(int n, int idx, float dx, float *dy, int incy, float *result) {
    // Use shared memory to reduce memory latency
    __shared__ float shared_dy[256]; // Assuming blockDim.x is 256
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop with stride and cache the data in shared memory
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        if (i >= idx && i % incy == 0) {
            shared_dy[threadIdx.x] = dy[i];
            __syncthreads();
            result[i] = shared_dy[threadIdx.x] - dx;
            __syncthreads();
        }
    }
}