#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void prod_strided_float(int n, int xOffset, float *dx, int incx, float result) {
    // Use shared memory for CUDA block reduction
    __shared__ float shared_result;
    int tid = threadIdx.x;
    
    // Initialize shared memory for each block
    if (tid == 0) shared_result = 1.0f;
    __syncthreads();

    // Use a local variable to accumulate product
    float thread_product = 1.0f;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        if(i >= xOffset && i % incx == 0) {
            thread_product *= dx[i];
        }
    }

    // Atomic multiplication to accumulate results in shared memory
    atomicExch(&shared_result, shared_result * thread_product);
    __syncthreads();

    // Write the block's final result to the global result
    if (tid == 0) {
        atomicExch(&result, result * shared_result);
    }
}