#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void sub_float(int n, float *a, float *b, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use loop to process more elements per thread to improve memory throughput
    for (; i < n; i += blockDim.x * gridDim.x)
    {
        sum[i] = a[i] - b[i];
    }
}