#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void add_float(int n, float *a, float *b, float *sum) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return; // Exit early if index is out of bounds
    sum[i] = a[i] + b[i]; // Perform addition
}