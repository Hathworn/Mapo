#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ceil_double(int n, int idx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int stride = blockDim.x * gridDim.x; // Stride for stepping through elements

    for (; i < n; i += stride) { // Loop over elements with stride
        if (i >= idx && i % incy == 0) // Check index conditions
            result[i] = ceil(dy[i]); // Perform ceiling operation
    }
}