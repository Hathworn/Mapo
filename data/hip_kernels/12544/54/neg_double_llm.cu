#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void neg_double(int n, int idx, double *dy, int incy, double *result) {
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop with stride of total threads
    while (i < n) {
        if(i >= idx && i % incy == 0) {
            result[i] = -dy[i]; // Negate and assign
        }
        i += blockDim.x * gridDim.x; // Move to next position
    }
}