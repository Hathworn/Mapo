#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void norm1_strided_float(int n, int xOffset, float *dx, int incx, float result) {
    // Use shared memory to reduce global memory accesses
    __shared__ float sharedSum[256]; 
    int tid = threadIdx.x; 
    sharedSum[tid] = 0.0f;

    // Use strided loop for better parallelism
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        if (i >= xOffset && i % incx == 0) {
            sharedSum[tid] += abs(dx[i]);
        }
    }

    // Synchronize threads in the block
    __syncthreads();

    // Reduce within block
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sharedSum[tid] += sharedSum[tid + s];
        }
        __syncthreads();
    }

    // Add the block result to the total result
    if (tid == 0) {
        atomicAdd(&result, sharedSum[0]);
    }
}