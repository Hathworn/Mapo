#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniform_double(int n, double lower, double upper, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation
    int totalThreads = gridDim.x * blockDim.x;

    while (i < n) {  // Use while instead of for-loop for clarity
        double u = result[i];
        result[i] = u * upper + (1 - u) * lower;
        i += totalThreads;  // Efficient index increment
    }
}