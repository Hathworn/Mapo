#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prod_strided_double(int n, int xOffset, double *dx, int incx, double *result) {
    __shared__ double sharedResult;
    if (threadIdx.x == 0) {
        sharedResult = 1.0; // Initialize shared memory
    }
    __syncthreads();

    double threadResult = 1.0; // Initialize thread-local result

    // Optimize loop for coalesced memory access and reduction
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        if (i >= xOffset && i % incx == 0) {
            threadResult *= dx[i];
        }
    }

    // Atomic multiplication to accumulate results in shared memory
    atomicExch(&sharedResult, sharedResult * threadResult); 
    __syncthreads();

    if (threadIdx.x == 0) {
        atomicExch(result, *result * sharedResult); // Update global result safely
    }
}