#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void pow_float(int n, int idx, float *dy, int incy, float raise, float *result) {
    // Cache grid and block dimensions for better readability
    int totalThreads = blockDim.x * gridDim.x;
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadIndex; i < n; i += totalThreads) {
        // Check if index meets conditions and apply pow operation
        if (i >= idx && i % incy == 0) {
            result[i] = powf(dy[i], raise);
        }
    }
}