#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

// Optimized global kernel function for floor_double
__global__ void floor_double(int n, int idx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = blockDim.x * gridDim.x;
    
    // Loop with grid-stride to cover all elements
    for (; i < n; i += gridStride) {
        // Check index condition and stride
        if (i >= idx && i % incy == 0) {
            result[i] = floor(dy[i]);
        }
    }
}