#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void ceil_double(int n, int idx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Optimize by unrolling the loop to reduce iterations
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = ceil(dy[i]);
        }
    }
}