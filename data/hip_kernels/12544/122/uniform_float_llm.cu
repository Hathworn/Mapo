#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniform_float(int n, float lower, float upper, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = gridDim.x * blockDim.x;
    
    // Use stride loop to ensure all elements are processed
    while (i < n) {
        float u = result[i];
        result[i] = u * upper + (1.0f - u) * lower; // Optimize: use 1.0f for float consistency
        i += totalThreads; // Process another element in the next iteration
    }
}