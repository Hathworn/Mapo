#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void floor_double(int n, int idx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use if statement outside of for loop for better readability and optimization
    if (i < n && i >= idx && i % incy == 0) {
        result[i] = floor(dy[i]);
    }
}