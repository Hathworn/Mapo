#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void floor_float(int n, int idx, float *dy, int incy, float *result) {
    // Calculate only once and reuse
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop unrolling for better performance
    for (; i + stride < n; i += stride * 2) {
        if (i >= idx && i % incy == 0) {
            result[i] = floor(dy[i]);
        }
        if (i + stride < n && (i + stride) >= idx && (i + stride) % incy == 0) {
            result[i + stride] = floor(dy[i + stride]);
        }
    }
    
    if (i < n && i >= idx && i % incy == 0) {
        result[i] = floor(dy[i]);
    }
}