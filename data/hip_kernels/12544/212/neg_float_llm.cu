#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void neg_float(int n, int idx, float *dy, int incy, float *result) {
    // Calculate the global index once
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a strided loop to cover all elements
    while (i < n) {
        if (i >= idx && i % incy == 0) {
            result[i] = -dy[i];
        }
        i += blockDim.x * gridDim.x;
    }
}