#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sub_strided_float(int n, int xOffset, int yOffset, float *dx, float *dy, int incx, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index
    int stride = blockDim.x * gridDim.x; // Compute stride for the grid
    for (; i < n; i += stride) {
        if (i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dy[i] - dx[i]; // Perform subtraction
        }
    }
}