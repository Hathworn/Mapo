#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void rdiv_scalar_double(int n, int idx, float dx, float *dy, int incy, double *result) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride-based loop for better global memory access pattern
    while (i < n) {
        // Check conditions and perform division
        if (i >= idx && i % incy == 0) {
            result[i] = dx / dy[i];
        }
        // Increment index by total number of threads in the grid
        i += blockDim.x * gridDim.x;
    }
}