#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void div_double(int n, double *a, double *b, double *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n && b[i] != 0) // Check divisor non-zero
    {
        sum[i] = a[i] / b[i];
    }
}