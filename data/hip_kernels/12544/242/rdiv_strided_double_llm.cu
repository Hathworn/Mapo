#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rdiv_strided_double(int n, int xOffset, int yOffset, double *dx, double *dy, int incx, int incy, double *result) {
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride to cover the entire data set
    int stride = blockDim.x * gridDim.x;

    // Loop over each element using the stride
    for (; i < n; i += stride) {
        // Check condition once here to minimize checks
        if(i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dx[i] - dy[i];
        }
    }
}