#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_strided_float(int n, int xOffset, int yOffset, float *dx, float *dy, int incx, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Iterate over array with stride
    for (; i < n; i += stride) {
        // Check conditions only when necessary
        if (i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dx[i] * dy[i];
        }
    }
}