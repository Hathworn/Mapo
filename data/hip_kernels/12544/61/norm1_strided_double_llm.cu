#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void norm1_strided_double(int n, int xOffset, double *dx, int incx, double *result) {
    __shared__ double shared_result;  // Use shared memory for intermediate results
    if (threadIdx.x == 0) shared_result = 0.0;  // Initialize shared memory
    __syncthreads();

    double local_sum = 0.0;  // Use local register for thread's partial sum

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        if (i >= xOffset && i % incx == 0)
            local_sum += abs(dx[i]);
    }

    atomicAdd(&shared_result, local_sum);  // Atomically add local sums to shared memory
    __syncthreads();

    if (threadIdx.x == 0)  // Single thread writes the result to global memory
        atomicAdd(result, shared_result);
}