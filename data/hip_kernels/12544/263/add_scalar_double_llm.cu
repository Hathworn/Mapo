#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_scalar_double(int n, int idx, double dx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use loop unrolling and guard clauses for better performance
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = dy[i] + dx;
        }
    }
}