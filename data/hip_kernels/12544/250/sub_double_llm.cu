#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sub_double(int n, double *a, double *b, double *sum)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use if statement without braces for single line
    if (i < n) sum[i] = a[i] - b[i];
}