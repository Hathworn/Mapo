#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void pow_double(int n, int idx, double *dy, int incy, double raise, double *result) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Iterate through elements with stride of total threads
    for (; i < n; i += blockDim.x * gridDim.x) {
        // Check conditions only once per loop iteration
        if (i >= idx && i % incy == 0) {
            result[i] = pow(dy[i], raise);
        }
    }
}