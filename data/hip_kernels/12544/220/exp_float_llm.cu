#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void exp_float(int n, int idx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int gridStride = blockDim.x * gridDim.x; // Calculate grid-stride to increase parallelism
    for (; i < n; i += gridStride) {
        if (i >= idx && i % incy == 0) {
            result[i] = cosf(dy[i]); // Efficient memory access within condition
        }
    }
}