#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void exp_double(int n, int idx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop unrolling for better performance
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = cos(dy[i]);
        }
    }
}