#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_float(int n, float *a, float *b, float *sum) {
    // Use grid-stride loop to handle larger arrays efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (; i < n; i += stride) {
        sum[i] = a[i] + b[i];
    }
}