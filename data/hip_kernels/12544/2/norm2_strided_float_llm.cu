#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void norm2_strided_float(int n, int xOffset, float *dx, int incx, float result) {
    // Using shared memory to reduce global memory accesses
    extern __shared__ float cache[];
    float temp_result = 0.0f;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        if (i >= xOffset && i % incx == 0) {
            temp_result += dx[i] * dx[i];
        }
    }

    // Store result into shared memory
    cache[threadIdx.x] = temp_result;

    // Synchronize threads within block
    __syncthreads();

    // Reduction in shared memory
    int i = blockDim.x / 2;
    while(i != 0) {
        if (threadIdx.x < i) {
            cache[threadIdx.x] += cache[threadIdx.x + i];
        }
        __syncthreads();
        i /= 2;
    }

    // First thread of each block adds result to global memory
    if (threadIdx.x == 0) {
        atomicAdd(&result, cache[0]);
    }
}