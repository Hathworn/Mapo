#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void sigmoid_float(int n, int idx, float *dy, int incy, float *result) {
  // Calculate unique global index
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  
  // Use stride pattern for enhanced parallelism
  int stride = blockDim.x * gridDim.x;
  
  for (; i < n; i += stride) {
    // Check condition to perform the operation
    if (i >= idx && i % incy == 0) {
      result[i] = 1.0f / (1.0f + expf(-dy[i]));
    }
  }
}