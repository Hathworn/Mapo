#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sub_scalar_double(int n, int idx, double dx, double *dy, int incy, double *result) {
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop to cover all elements
    int stride = blockDim.x * gridDim.x;
    
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = dy[i] - dx;
        }
    }
}