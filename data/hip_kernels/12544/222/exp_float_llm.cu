#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void exp_float(int n, int idx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use strided loop for better occupancy
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = expf(dy[i]);
        }
    }
}