#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void rdiv_strided_float(int n,int xOffset,int yOffset, float *dx,float *dy,int incx,int incy,float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride loop to efficiently cover all elements
    int stride = blockDim.x * gridDim.x;
    for (; i < n; i += stride) {
        // Simplify condition checking by incorporating offset directly
        if ((i >= xOffset) && (i >= yOffset) && ((i - xOffset) % incx == 0) && ((i - yOffset) % incy == 0)) {
            result[i] = dx[i] / dy[i];
        }
    }
}