#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void sqrt_double(int n, int idx, double *dy, int incy, double *result) {
    // Use shared memory to reduce global memory access if needed
    __shared__ double shared_dy[256]; // Assuming block size of 256, adjust if necessary
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    while (i < n) {
        if (i >= idx && i % incy == 0) {
            // Load data to shared memory, then compute
            shared_dy[threadIdx.x] = dy[i];
            __syncthreads();

            result[i] = sqrt(shared_dy[threadIdx.x]);
        }
        i += blockDim.x * gridDim.x;
    }
}