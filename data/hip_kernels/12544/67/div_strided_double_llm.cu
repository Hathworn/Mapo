#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void div_strided_double(int n, int xOffset, int yOffset, double *dx, double *dy, int incx, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global index
    int stride = blockDim.x * gridDim.x; // Compute the stride
    for (; i < n; i += stride) {
        if (i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dy[i] / dx[i]; // Perform division
        }
    }
}