#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void div_strided_double(int n, int xOffset, int yOffset, double *dx, double *dy, int incx, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop with stride over the grid to cover all elements
    while (i < n) {
        // Check conditions to ensure valid indices before division
        if (i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dy[i] / dx[i];
        }
        // Increment index by total number of threads in the grid for next stride
        i += blockDim.x * gridDim.x;
    }
}