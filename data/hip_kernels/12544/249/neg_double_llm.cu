#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void neg_double(int n, int idx, double *dy, int incy, double *result) {
    // Calculate the index for this thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds and meets condition
    if (i < n && i >= idx && i % incy == 0) {
        result[i] = -dy[i]; // Negate value
    }
}