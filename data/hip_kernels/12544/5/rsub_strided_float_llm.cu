#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void rsub_strided_float(int n, int xOffset, int yOffset, float *dx, float *dy, int incx, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate the global thread index
    int step = blockDim.x * gridDim.x;  // Calculate the step size

    while (i < n) {
        // Process elements only if they meet the conditions
        if (i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dx[i] - dy[i];
        }
        i += step;  // Move to the next set of elements
    }
}