#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_scalar_double(int n, int idx, double dx, double *dy, int incy, double *result) {
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a `while` loop to prevent repeated condition checking
    while (i < n) {
        if (i >= idx && i % incy == 0) {
            result[i] = dy[i] * dx;
        }
        i += blockDim.x * gridDim.x; // Increment using grid-stride loop
    }
}