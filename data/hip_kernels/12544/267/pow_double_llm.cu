#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_double(int n, int idx, double *dy, int incy, double raise, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global thread index
    int stride = blockDim.x * gridDim.x; // Calculate the stride

    while (i < n) { // Use a while loop for efficient iteration over data elements
        if (i >= idx && i % incy == 0) {
            result[i] = pow(dy[i], raise); // Only compute power if the condition is met
        }
        i += stride; // Increment by stride to cover all elements
    }
}