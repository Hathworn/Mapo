#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void add_strided_float(int n, int xOffset, int yOffset, float *dx, float *dy, int incx, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index

    // Use loop unrolling and simplify conditions for better performance
    while (i < n) {
        if (i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dy[i] + dx[i];  // Perform addition
        }
        i += blockDim.x * gridDim.x;  // Stride the index
    }
}