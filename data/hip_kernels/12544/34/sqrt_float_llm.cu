#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void sqrt_float(int n, int idx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int stride = blockDim.x * gridDim.x; // Calculate stride for grid
    
    for (; i < n; i += stride) { // Use stride-based iteration
        if (i >= idx && i % incy == 0) {
            result[i] = sqrtf(dy[i]); // Compute square root
        }
    }
}