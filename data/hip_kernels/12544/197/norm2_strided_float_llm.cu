#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void norm2_strided_float(int n, int xOffset, float *dx, int incx, float result) {
    // Use shared memory to accumulate results for optimal parallel reduction
    extern __shared__ float shared_result[];
    int tid = threadIdx.x;
    shared_result[tid] = 0.0;

    for (int i = blockIdx.x * blockDim.x + tid; i < n; i += blockDim.x * gridDim.x) {
        if (i >= xOffset && (i - xOffset) % incx == 0) {
            shared_result[tid] += powf(dx[i], 2);
        }
    }

    __syncthreads();

    // Perform reduction within each block
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            shared_result[tid] += shared_result[tid + stride];
        }
        __syncthreads();
    }

    // Accumulate result from this block
    if (tid == 0) {
        atomicAdd(&result, shared_result[0]);
    }
}