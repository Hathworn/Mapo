#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void eq_strided_double(int n, int xOffset, int yOffset, double *dx, double *dy, int incx, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop with striding
    for (; i < n; i += stride) {
        if (i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dy[i] == dx[i];
        }
    }
}