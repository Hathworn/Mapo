#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void log_double(int n, int idx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use stride-based loop for better parallel efficiency
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = log(dy[i]);
        }
    }
}