#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rsub_strided_double(int n, int xOffset, int yOffset, double *dx, double *dy, int incx, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Loop unrolling and avoid recalculating indexes
    int stride = blockDim.x * gridDim.x;
    while (i < n) {
        if(i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0)
            result[i] = dx[i] / dy[i];
        i += stride;
    }
}