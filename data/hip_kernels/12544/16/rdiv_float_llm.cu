#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void rdiv_float(int n, float *a, float *b, float *sum)
{
    // Use grid stride loop for optimized parallel execution
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int idx = i; idx < n; idx += stride) {
        sum[idx] = b[idx] / a[idx];
    }
}