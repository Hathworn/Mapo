#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sigmoid_float(int n, int idx, float *dy, int incy, float *result) {
    // Use thread ID to calculate the index directly
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = blockDim.x * gridDim.x;

    while (i < n) {
        if (i >= idx && i % incy == 0) {
            result[i] = 1.0f / (1.0f + expf(-dy[i]));
        }
        i += totalThreads;
    }
}