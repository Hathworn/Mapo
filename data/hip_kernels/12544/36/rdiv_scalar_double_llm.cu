#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void rdiv_scalar_double(int n, int idx, float dx, float *dy, int incy, double *result) {
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use strided loop for processing elements
    for (; i < n; i += blockDim.x * gridDim.x) {
        // Only compute for valid elements
        if (i >= idx && i % incy == 0) {
            result[i] = dx / dy[i];
        }
    }
}