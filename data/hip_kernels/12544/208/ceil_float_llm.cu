#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ceil_float(int n, int idx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global index
    int stride = blockDim.x * gridDim.x; // Calculate the stride for grid-stride loop

    // Optimize with a grid-stride loop
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = ceil(dy[i]); // Apply ceil function
        }
    }
}