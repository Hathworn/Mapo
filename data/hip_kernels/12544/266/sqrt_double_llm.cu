#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sqrt_double(int n, int idx, double *dy, int incy, double *result) {
    // Cache thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;

    // Loop unrolling and grid-stride loop
    while (i < n) {
        if (i >= idx && i % incy == 0) {
            result[i] = sqrt(dy[i]);
        }
        i += gridSize;
    }
}