#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rsub_double(int n, double *a, double *b, double *sum)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Stride loop to allow handling arrays larger than grid size
    for (int index = i; index < n; index += blockDim.x * gridDim.x)
    {
        sum[index] = b[index] - a[index];
    }
}