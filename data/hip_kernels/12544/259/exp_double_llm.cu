#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void exp_double(int n, int idx, double *dy, int incy, double *result) {
    // Calculate global index for the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a stride loop to cover all elements
    int stride = blockDim.x * gridDim.x;

    while (i < n) {
        // Check conditions and perform computation
        if (i >= idx && i % incy == 0) {
            result[i] = exp(dy[i]);
        }
        i += stride; // Move to next element in the stride loop
    }
}