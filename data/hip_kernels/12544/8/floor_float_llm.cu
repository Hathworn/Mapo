#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void floor_float(int n, int idx, float *dy, int incy, float *result) {
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure all threads exit if exceeding the number of elements to process
    if (i < n && i >= idx && i % incy == 0) {
        // Perform the floor operation if within bounds and meets criteria
        result[i] = floorf(dy[i]);  // Use floorf for float for better performance
    }
}