#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void abs_float(int n, int idx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop with stride for better performance
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = fabsf(dy[i]); // Use fabsf for float type
        }
    }
}