#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void norm1_strided_float(int n, int xOffset, float *dx, int incx, float *result) {
    // Use shared memory to accumulate results for each block
    __shared__ float shared_sum[256];
    int tid = threadIdx.x;
    shared_sum[tid] = 0;

    // Strided loop modified for better memory access patterns
    for (int i = blockIdx.x * blockDim.x + tid; i < n; i += blockDim.x * gridDim.x) {
        if (i >= xOffset && i % incx == 0) {
            shared_sum[tid] += fabsf(dx[i]);
        }
    }

    // Synchronize threads within the block
    __syncthreads();

    // Reduce partial results within the block
    if (tid < 128) shared_sum[tid] += shared_sum[tid + 128];
    __syncthreads();
    if (tid < 64) shared_sum[tid] += shared_sum[tid + 64];
    __syncthreads();

    // Unrolling the final warp for efficiency
    if (tid < 32) {
        shared_sum[tid] += shared_sum[tid + 32];
        shared_sum[tid] += shared_sum[tid + 16];
        shared_sum[tid] += shared_sum[tid + 8];
        shared_sum[tid] += shared_sum[tid + 4];
        shared_sum[tid] += shared_sum[tid + 2];
        shared_sum[tid] += shared_sum[tid + 1];
    }

    // Add block result to global result
    if (tid == 0) atomicAdd(result, shared_sum[0]);
}