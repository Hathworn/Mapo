#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_float(int n, int idx, float *dy, int incy, float raise, float *result) {
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate total stride
    
    for (; i < n; i += stride) { // Utilize stride loop to cover all elements
        if (i >= idx && i % incy == 0) // Ensure conditions are met before computation
            result[i] = powf(dy[i], raise); // Perform the powf operation
    }
}