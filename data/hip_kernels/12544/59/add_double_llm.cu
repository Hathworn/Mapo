#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void add_double(int n, double *a, double *b, double *sum) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Add stride for grid-stride loop
    for (; i < n; i += stride) { // Use grid-stride loop for better utilization
        sum[i] = a[i] + b[i];
    }
}