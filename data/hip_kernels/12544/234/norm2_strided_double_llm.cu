#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void norm2_strided_double(int n, int xOffset, double *dx, int incx, double *result) {
    // Use shared memory to accumulate results from threads
    __shared__ double sharedResult[256];
    int tid = threadIdx.x;
    sharedResult[tid] = 0.0;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        if (i >= xOffset && i % incx == 0) {
            sharedResult[tid] += dx[i] * dx[i];
        }
    }

    // Synchronize threads before reduction
    __syncthreads();

    // Reduce within the block to calc block partial sum
    if (tid < 128) sharedResult[tid] += sharedResult[tid + 128];
    __syncthreads();
    if (tid < 64) sharedResult[tid] += sharedResult[tid + 64];
    __syncthreads();
    if (tid < 32) {
        sharedResult[tid] += sharedResult[tid + 32];
        sharedResult[tid] += sharedResult[tid + 16];
        sharedResult[tid] += sharedResult[tid + 8];
        sharedResult[tid] += sharedResult[tid + 4];
        sharedResult[tid] += sharedResult[tid + 2];
        sharedResult[tid] += sharedResult[tid + 1];
    }

    // Write block result to global memory
    if (tid == 0) atomicAdd(result, sharedResult[0]);
}