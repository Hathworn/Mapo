#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void norm2_strided_double(int n, int xOffset, double *dx, int incx, double* result) {
    extern __shared__ double shared_data[];

    int tid = threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;
    double sum = 0.0;  // Use a register for local accumulation

    // Iterate over elements with striding
    for (int i = blockIdx.x * blockDim.x + tid; i < n; i += gridSize) {
        if(i >= xOffset && i % incx == 0) {
            sum += pow(dx[i], 2);
        }
    }

    shared_data[tid] = sum; // Store local sum in shared memory
    __syncthreads();

    // Perform reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();
    }

    // First thread writes block's accumulated result to global memory
    if (tid == 0) {
        atomicAdd(result, shared_data[0]);
    }
}