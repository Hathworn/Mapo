#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void rsub_scalar_double(int n, int idx,double dx,double *dy,int incy,double *result) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread operates within array bounds
    if (i < n && i >= idx && i % incy == 0) {
        result[i] = dx - dy[i]; // Perform subtraction
    }
}