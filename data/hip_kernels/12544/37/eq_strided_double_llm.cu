#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void eq_strided_double(int n, int xOffset, int yOffset, double *dx, double *dy, int incx, int incy, double *result) {
    // Calculate the global index based on block and thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Stride through output array for all threads collectively
    int stride = blockDim.x * gridDim.x;

    // Process each element with stride loop
    for (; i < n; i += stride) {
        // Check bounds and strides, then compute results
        if (i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dy[i] == dx[i];
        }
    }
}