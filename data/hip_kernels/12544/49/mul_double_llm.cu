#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void mul_double(int n, double *a, double *b, double *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // Perform multiply in registers
        double temp_a = a[i];
        double temp_b = b[i];
        sum[i] = temp_a * temp_b;
    }
}