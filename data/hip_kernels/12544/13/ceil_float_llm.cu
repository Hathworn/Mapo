#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void ceil_float(int n, int idx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int stride = blockDim.x * gridDim.x; // Calculate stride for all threads
    for (; i < n; i += stride) { 
        if (i >= idx && i % incy == 0) {
            result[i] = ceilf(dy[i]); // Use ceilf for float type
        }
    }
}