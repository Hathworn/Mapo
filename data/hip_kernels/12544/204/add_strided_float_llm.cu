#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_strided_float(int n, int xOffset, int yOffset, float *dx, float *dy, int incx, int incy, float *result) {
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop to handle array bounds
    int stride = blockDim.x * gridDim.x;
    
    // Loop with improved boundary checks
    for (; i < n; i += stride) {
        if (i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dy[i] + dx[i];
        }
    }
}