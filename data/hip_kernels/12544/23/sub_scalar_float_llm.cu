#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void sub_scalar_float(int n, int idx, float dx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Unroll the loop to improve performance
    for (int j = i; j < n; j += stride) {
        if(j >= idx && j % incy == 0)
            result[j] = dy[j] - dx;
    }
}