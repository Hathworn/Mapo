#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void tanh_double(int n, int idx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Streamlined loop condition for efficiency
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            // Avoid redundant calculations
            double val = dy[i];
            result[i] = tanh(val);
        }
    }
}