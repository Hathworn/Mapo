#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void sigmoid_double(int n, int idx, double *dy, int incy, double *result) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use strided loop for better memory access
    for (; i < n; i += blockDim.x * gridDim.x) {
        // Check conditions and compute sigmoid
        if(i >= idx && i % incy == 0)
            result[i] = 1.0 / (1.0 + exp(-dy[i]));
    }
}