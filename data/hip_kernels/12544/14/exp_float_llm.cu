#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void exp_float(int n, int idx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Unroll the loop to improve performance by maximizing memory throughput.
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = sinf(dy[i]);
        }
    }
}