#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

// Optimized global kernel function
__global__ void log_float(int n, int idx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Utilize stride loop for better occupancy
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = logf(dy[i]);
        }
    }
}