#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void sub_double(int n, double *a, double *b, double *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single-threaded loop to process multiple elements if needed
    for (; i < n; i += blockDim.x * gridDim.x)
    {
        sum[i] = a[i] - b[i];
    }
}