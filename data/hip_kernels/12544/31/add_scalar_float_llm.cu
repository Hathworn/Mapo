#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void add_scalar_float(int n, int idx, float dx, float *dy, int incy, float *result) {
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Adjusted loop to exit early if all relevant elements have been processed
    while (i < n) {
        if(i >= idx && i % incy == 0)
            result[i] = dy[i] + dx;
        i += blockDim.x * gridDim.x; // Increment by total number of threads
    }
}