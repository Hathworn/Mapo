#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sub_strided_double(int n, int xOffset, int yOffset, double *dx, double *dy, int incx, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a loop stride that matches the total number of threads
    int stride = blockDim.x * gridDim.x;

    // Loop through the array with the proper stride
    for (; i < n; i += stride) {
        if (i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dy[i] - dx[i];
        }
    }
}