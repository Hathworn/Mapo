#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sign_float(int n, int idx, float *dy, int incy, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Unrolled loop to optimize memory access and improve performance
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            float x = dy[i];
            result[i] = (x > 0) - (x < 0);
        }
    }
}