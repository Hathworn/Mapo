#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sub_float(int n, float *a, float *b, float *sum)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for better global memory access pattern
    for (; i < n; i += blockDim.x * gridDim.x)
    {
        sum[i] = a[i] - b[i];
    }
}