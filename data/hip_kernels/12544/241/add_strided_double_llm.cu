#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void add_strided_double(int n, int xOffset, int yOffset, double *dx, double *dy, int incx, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Use strided loop for optimal memory access
    for (; i < n; i += stride) {
        if (i >= xOffset && i >= yOffset && i % incx == 0 && i % incy == 0) {
            result[i] = dy[i] + dx[i];
        }
    }
}