#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void norm1_strided_double(int n, int xOffset, double *dx, int incx, double *result) {
    // Create shared memory to reduce atomic operations overhead
    extern __shared__ double sharedResult[];
    int tid = threadIdx.x;
    sharedResult[tid] = 0.0;

    // Traverse elements with strided pattern
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        if (i >= xOffset && i % incx == 0) {
            sharedResult[tid] += abs(dx[i]);
        }
    }

    // Synchronize threads within the block
    __syncthreads();

    // Use parallel reduction to sum results within a block
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedResult[tid] += sharedResult[tid + stride];
        }
        __syncthreads();
    }

    // Update global result with result from block 0 thread 0
    if (tid == 0) {
        atomicAdd(result, sharedResult[0]);
    }
}