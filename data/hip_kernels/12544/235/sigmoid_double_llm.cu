#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sigmoid_double(int n, int idx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Iterate with a stride, optimizing loop increments
    while (i < n) {
        if (i >= idx && i % incy == 0) {
            result[i] = 1.0 / (1.0 + exp(-dy[i]));
        }
        i += stride;
    }
}