#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void sub_scalar_double(int n, int idx, double dx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Efficient loop with stride to maximize occupancy
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = dy[i] - dx;
        }
    }
}