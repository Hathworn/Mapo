#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void exp_double(int n, int idx, double *dy, int incy, double *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Unrolling loop for better performance
    for (; i < n; i += stride) {
        if (i >= idx && i % incy == 0) {
            result[i] = cos(dy[i]);
        }
    }
}