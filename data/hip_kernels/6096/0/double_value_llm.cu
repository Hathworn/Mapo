#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void double_value(double *x, double *y)
{
    // Calculate global thread index for handling larger problem sizes
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Perform the operation using the calculated global index
    y[idx] = 2. * x[idx];
}