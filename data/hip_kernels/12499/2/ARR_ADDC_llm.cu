#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ARR_ADDC(float* result, float* in1, float* in2, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for larger data
    for (int i = index; i < N; i += stride) // Use loop to cover entire data
    {
        result[i] = in1[i] + in2[i];
    }
}