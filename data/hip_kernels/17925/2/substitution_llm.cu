#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Helper function for using CUDA to call kernel functions
hipError_t cuda_code(float* , float*, int , int );
__device__ float sum = 0;

__global__ void substitution(int i, int N, float *row, float *matrix, float* resultVector) {
    int j = i + blockIdx.x * blockDim.x + threadIdx.x;

    if (j > i && j < N) {
        int ij = j + (N + 1) * i;
        float temp = matrix[ij] * resultVector[j];

        atomicAdd(&sum, temp);
        row[j] = temp;
    }
    // __syncthreads() not needed here, since 'sum' is a single global variable and atomicAdd is used.
}