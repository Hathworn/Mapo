#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void eladd(float * inA, float * inB, int length)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Use grid-stride loop to handle more elements beyond a single block
    for (; idx < length; idx += blockDim.x * gridDim.x)
    {
        inA[idx] += inB[idx];
    }
}