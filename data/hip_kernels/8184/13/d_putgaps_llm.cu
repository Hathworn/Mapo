#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void d_putgaps(float *sne7, float *snaw, int *aw2ali, const int snno)
{
    // Compute sino index using blockDim.x for improved readability
    int sni = threadIdx.x + blockIdx.y * blockDim.x;

    // Sino bin index remains unchanged
    int awi = blockIdx.x;

    // Ensuring we are within bounds before assignment
    if (sni < snno) {
        // Use calculated index to set value in sne7 array efficiently
        sne7[aw2ali[awi] * snno + sni] = snaw[awi * snno + sni];
    }
}