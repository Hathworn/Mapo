#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rdiv(float * res, const unsigned int * fsum, const float * csum) {
    // Calculate unique global thread index directly
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // Check if within bounds to prevent accessing out of bounds memory
    if (idx < N) {
        res[idx] = (float)fsum[idx] / csum[idx];
    }
}