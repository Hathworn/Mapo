#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void eldiv0(float *inA, float *inB, int length)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        // Use registers to hold values for faster operations
        float valA = inA[idx];
        float valB = inB[idx];
        
        inA[idx] = valA / valB;
    }
}