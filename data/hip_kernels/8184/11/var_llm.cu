#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void var(float * __restrict__ M1, float * __restrict__ M2, float * __restrict__ X, int b, size_t nele) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nele) {
        // Cache output of index to registers to reduce repetitive memory fetch
        float m1 = M1[idx];
        float x = X[idx];

        float delta = x - m1;
        m1 += delta / (b + 1);

        // Store only once to global memory
        M1[idx] = m1;
        M2[idx] += delta * (x - m1);
    }
}