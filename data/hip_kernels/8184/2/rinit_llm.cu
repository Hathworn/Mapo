#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rinit(float *init, const unsigned int *fsum, const float *ncrs) {
    // Calculate global thread index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use fast-math intrinsic for square root to improve performance
    init[idx] = __fsqrt_rn((float)fsum[idx] / ncrs[idx]);
}