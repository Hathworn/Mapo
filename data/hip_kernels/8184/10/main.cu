#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "d_sct_axinterp.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *sct3d = NULL;
hipMalloc(&sct3d, XSIZE*YSIZE);
const float *scts1 = NULL;
hipMalloc(&scts1, XSIZE*YSIZE);
const int4 *sctaxR = NULL;
hipMalloc(&sctaxR, XSIZE*YSIZE);
const float4 *sctaxW = NULL;
hipMalloc(&sctaxW, XSIZE*YSIZE);
const short *sn1_sn11 = NULL;
hipMalloc(&sn1_sn11, XSIZE*YSIZE);
int NBIN = 1;
int NSN1 = 1;
int SPN = 1;
int offtof = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
d_sct_axinterp<<<gridBlock, threadBlock>>>(sct3d,scts1,sctaxR,sctaxW,sn1_sn11,NBIN,NSN1,SPN,offtof);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
d_sct_axinterp<<<gridBlock, threadBlock>>>(sct3d,scts1,sctaxR,sctaxW,sn1_sn11,NBIN,NSN1,SPN,offtof);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
d_sct_axinterp<<<gridBlock, threadBlock>>>(sct3d,scts1,sctaxR,sctaxW,sn1_sn11,NBIN,NSN1,SPN,offtof);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}