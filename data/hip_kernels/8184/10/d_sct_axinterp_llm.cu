#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void d_sct_axinterp(float *sct3d, const float *scts1, const int4 *sctaxR, const float4 *sctaxW, const short *sn1_sn11, int NBIN, int NSN1, int SPN, int offtof)
{
    // Use local variables and registers for optimization
    char ics = threadIdx.x;
    char icu = 2 * threadIdx.y;
    short sni = blockIdx.x;

    float tmp1 = 0.0f, tmp2 = 0.0f;
    int baseIndex = NBIN * blockDim.x;
    int offset1 = icu * blockDim.x + ics;
    int offset2 = (icu + 1) * blockDim.x + ics;

    // Precompute indices for better memory access patterns
    int index1 = baseIndex * sctaxR[sni].x + offset1;
    int index2 = baseIndex * sctaxR[sni].y + offset1;
    int index3 = baseIndex * sctaxR[sni].z + offset1;
    int index4 = baseIndex * sctaxR[sni].w + offset1;

    int index1_2 = baseIndex * sctaxR[sni].x + offset2;
    int index2_2 = baseIndex * sctaxR[sni].y + offset2;
    int index3_2 = baseIndex * sctaxR[sni].z + offset2;
    int index4_2 = baseIndex * sctaxR[sni].w + offset2;

    // Use Fused Multiply-Add for improved performance
    tmp1 = fmaf(sctaxW[sni].x, scts1[index1], tmp1);
    tmp1 = fmaf(sctaxW[sni].y, scts1[index2], tmp1);
    tmp1 = fmaf(sctaxW[sni].z, scts1[index3], tmp1);
    tmp1 = fmaf(sctaxW[sni].w, scts1[index4], tmp1);

    tmp2 = fmaf(sctaxW[sni].x, scts1[index1_2], tmp2);
    tmp2 = fmaf(sctaxW[sni].y, scts1[index2_2], tmp2);
    tmp2 = fmaf(sctaxW[sni].z, scts1[index3_2], tmp2);
    tmp2 = fmaf(sctaxW[sni].w, scts1[index4_2], tmp2);

    // Conditional logic minimized for coherence
    int outputIndex1 = offtof + sni * NBIN + offset1;
    int outputIndex2 = offtof + sni * NBIN + offset2;

    if (SPN == 1) {
        sct3d[outputIndex1] = tmp1;
        sct3d[outputIndex2] = tmp2;
    }
    else if (SPN == 11 && sni < NSN1) {
        short sni11 = sn1_sn11[sni];
        int sni11Index1 = offtof + sni11 * NBIN + offset1;
        int sni11Index2 = offtof + sni11 * NBIN + offset2;
        atomicAdd(sct3d + sni11Index1, tmp1);
        atomicAdd(sct3d + sni11Index2, tmp2);
    }
}