#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void radd(float * resp, const float * res, float alpha) {
    // Calculate global index using 2D grid and block for flexibility
    int idx = threadIdx.x + blockIdx.x * blockDim.x + (blockIdx.y * gridDim.x + blockIdx.y) * blockDim.x * blockDim.y;

    // Check if index is within bounds before accessing arrays
    if (idx < gridDim.x * blockDim.x * blockDim.y) {
        // Perform the computation
        resp[idx] = (1 - alpha) * resp[idx] + alpha * res[idx];
    }
}