#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sneladd(float *inA, float *inB, int *sub, int Nprj, int snno)
{
    int idz = threadIdx.x + blockDim.x * blockIdx.x;
    int projIdx = blockIdx.y; // Cache blockIdx.y to avoid repeated access
    
    if (projIdx < Nprj && idz < snno) {
        int indexA = snno * projIdx + idz; // Calculate index for inA
        int indexB = snno * sub[projIdx] + idz; // Calculate index for inB
        inA[indexA] += inB[indexB];
    }
}