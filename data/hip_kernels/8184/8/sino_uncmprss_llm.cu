#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sino_uncmprss(unsigned int * dsino, unsigned char * p1sino, unsigned char * d1sino, int ifrm, int nele)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < nele) {
        unsigned int value = dsino[ifrm * nele + idx]; // Cache the repetitive array access
        d1sino[2 * idx] = (unsigned char)((value >> 8) & 0x000000ff);
        d1sino[2 * idx + 1] = (unsigned char)((value >> 24) & 0x000000ff);
        p1sino[2 * idx] = (unsigned char)(value & 0x000000ff);
        p1sino[2 * idx + 1] = (unsigned char)((value >> 16) & 0x000000ff);
    }
}