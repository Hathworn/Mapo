#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elmsk(float *inA, float *inB, bool *msk, int length)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    // Unroll the loop to process two elements at a time
    for (; idx + 1 < length; idx += blockDim.x * gridDim.x * 2) {
        if (msk[idx]) 
            inA[idx] *= inB[idx];
        else 
            inA[idx] = 0;
        
        if (msk[idx + 1]) 
            inA[idx + 1] *= inB[idx + 1];
        else 
            inA[idx + 1] = 0;
    }

    // Handle the remaining element if length is odd
    if (idx < length) {
        if (msk[idx]) 
            inA[idx] *= inB[idx];
        else 
            inA[idx] = 0;
    }
}