#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved global kernel function
__global__ void elmult(float *inA, float *inB, int length)
{
    // Calculate unique global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid-stride loop for better performance
    for (int i = idx; i < length; i += blockDim.x * gridDim.x) {
        inA[i] *= inB[i];
    }
}