#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numCols, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    // Improved index calculation to prevent out-of-bound access
    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Efficient memory access within valid range
    if (eidx < numElements) {
        for (uint idx = eidx; idx < numElements; idx += numCols) {
            sum += a[idx] * b[idx];
        }
    }

    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Unroll reduction with warp-level convergence
    for (uint stride = DP_BLOCKSIZE / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write block result to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}