#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Calculate y and x using integer division and modulus for efficiency
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        // Use integer modulus to map target indices to source indices
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Directly index source and assign to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}