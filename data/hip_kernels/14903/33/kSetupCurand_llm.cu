#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate global thread index
    const uint tidx = blockIdx.x * blockDim.x + threadIdx.x;
    /* Each thread gets the same seed, a different sequence number,
       no offset */
    hiprand_init(seed, tidx, 0, &state[tidx]);
}