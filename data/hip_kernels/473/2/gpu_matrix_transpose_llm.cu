#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16

__global__ void gpu_matrix_transpose(int* mat_in, int* mat_out, unsigned int rows, unsigned int cols)
{
    // Calculate transposed index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure indices are within bounds
    if (idx < cols && idy < rows)
    {
        unsigned int pos = idy * cols + idx;
        unsigned int trans_pos = idx * rows + idy;
        mat_out[trans_pos] = mat_in[pos]; // Assign transposed values
    }
}