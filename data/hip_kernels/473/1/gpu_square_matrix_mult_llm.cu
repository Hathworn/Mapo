#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16

__global__ void gpu_square_matrix_mult(int *d_a, int *d_b, int *d_result, int n)
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;

    for (int sub = 0; sub < gridDim.x; ++sub)
    {
        int tile_a_idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        int tile_b_idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;

        // Load tiles into shared memory, zero-pad if out-of-bounds
        tile_a[threadIdx.y][threadIdx.x] = (tile_a_idx < n * n) ? d_a[tile_a_idx] : 0;
        tile_b[threadIdx.y][threadIdx.x] = (tile_b_idx < n * n) ? d_b[tile_b_idx] : 0;
        __syncthreads();

        // Matrix multiplication within block
        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}