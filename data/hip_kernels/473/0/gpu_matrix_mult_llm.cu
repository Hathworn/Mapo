#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16

__global__ void gpu_matrix_mult(int *a, int *b, int *c, int m, int n, int k)
{
    // Calculate global thread row and column indices
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize shared memory for tiles of matrices a and b
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int sum = 0; 

    // Loop through tiles of matrix
    for (int tileIdx = 0; tileIdx < (n + BLOCK_SIZE - 1) / BLOCK_SIZE; ++tileIdx) {
        
        // Load data into shared memory tiles
        if (row < m && tileIdx * BLOCK_SIZE + threadIdx.x < n)
            tile_a[threadIdx.y][threadIdx.x] = a[row * n + tileIdx * BLOCK_SIZE + threadIdx.x];
        else
            tile_a[threadIdx.y][threadIdx.x] = 0;

        if (col < k && tileIdx * BLOCK_SIZE + threadIdx.y < n)
            tile_b[threadIdx.y][threadIdx.x] = b[(tileIdx * BLOCK_SIZE + threadIdx.y) * k + col];
        else
            tile_b[threadIdx.y][threadIdx.x] = 0;

        __syncthreads();  // Synchronize threads to ensure complete loading

        // Compute partial results
        for (int j = 0; j < BLOCK_SIZE; ++j) {
            sum += tile_a[threadIdx.y][j] * tile_b[j][threadIdx.x];
        }
        
        __syncthreads();  // Synchronize before loading new tile
    }

    // Write back result to matrix c
    if (row < m && col < k) {
        c[row * k + col] = sum;
    }
}