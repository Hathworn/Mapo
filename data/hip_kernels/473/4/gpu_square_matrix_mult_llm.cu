#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_square_matrix_mult(int *d_a, int *d_b, int *d_result, int n)
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;

    // Iterate over all sub-matrices ('tiles')
    for (int sub = 0; sub < gridDim.x; ++sub)
    {
        // Load data from global memory to shared memory with boundary checks
        int idx_a = row * n + sub * BLOCK_SIZE + threadIdx.x;
        tile_a[threadIdx.y][threadIdx.x] = (idx_a < n * n) ? d_a[idx_a] : 0;

        int idx_b = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        tile_b[threadIdx.y][threadIdx.x] = (idx_b < n * n) ? d_b[idx_b] : 0;

        // Synchronize to ensure all data is loaded
        __syncthreads();

        // Perform multiplication on the loaded tiles
        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }

        // Synchronize before loading new subtile
        __syncthreads();
    }

    // Write result to global memory
    if(row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}