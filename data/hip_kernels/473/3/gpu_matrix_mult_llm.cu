#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_matrix_mult(int *a, int *b, int *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize sum to accumulate results
    int sum = 0;
    
    // Ensure within matrix bounds
    if (col < k && row < m)
    {
        // Loop unrolling for efficient computation
        for (int i = 0; i < n; i += 4)
        {
            sum += a[row * n + i] * b[i * k + col];
            if (i + 1 < n)
                sum += a[row * n + i + 1] * b[(i + 1) * k + col];
            if (i + 2 < n)
                sum += a[row * n + i + 2] * b[(i + 2) * k + col];
            if (i + 3 < n)
                sum += a[row * n + i + 3] * b[(i + 3) * k + col];
        }
        
        // Store result in the result matrix
        c[row * k + col] = sum;
    }
}