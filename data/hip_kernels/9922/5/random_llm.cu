#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void random(double *x, hiprandState *global_state) {
    // Optimize thread indexing for potential grid-stride loop
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_size = blockDim.x * gridDim.x;
    
    // Loop through grid-stride to allow more work per thread block
    for (int i = tid; i < grid_size; i += grid_size) {
        hiprandState local_state = global_state[i]; // Use local register for state
        x[i] = __double2hiint(hiprand_double(&local_state)); // Ensure precision by using int conversion
        global_state[i] = local_state; // Store local state back to global
    }
}