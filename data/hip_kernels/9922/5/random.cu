#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void random(double *x, hiprandState *global_state){
int tid =  blockIdx.x;
hiprandState local_state;
local_state = global_state[tid];
x[tid] = (double) hiprand(&local_state);
global_state[tid] = local_state;
}