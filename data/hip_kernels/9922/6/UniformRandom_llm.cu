#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UniformRandom(double *x, hiprandState *global_state) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (tid < gridDim.x * blockDim.x) { // Check boundary to ensure valid access
        hiprandState local_state = global_state[tid]; // Directly access global_state
        x[tid] = (double)hiprand_uniform(&local_state);
        global_state[tid] = local_state; // Store updated state back
    }
}