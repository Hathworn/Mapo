#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_random(unsigned long long *seed, hiprandState  *global_state) {
    // Using threadIdx.x for parallel processing within blocks
    int tid = blockIdx.x * blockDim.x + threadIdx.x; 
    unsigned long long local_seed = seed[tid];
    hiprandState local_state;
    hiprand_init(local_seed, tid, 0, &local_state);
    global_state[tid] = local_state;
}