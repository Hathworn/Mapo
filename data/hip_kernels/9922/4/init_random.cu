#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_random(unsigned long long *seed, hiprandState  *global_state){
int tid = blockIdx.x;
unsigned long long local_seed = seed[tid];
hiprandState local_state;
local_state = global_state[tid];
hiprand_init(local_seed,tid,0, &local_state);
global_state[tid] = local_state;
}