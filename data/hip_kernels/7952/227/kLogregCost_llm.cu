#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        if (labelp != maxp) {
            correctProbs[tx] = 0.0f;
        } else {
            int numMax = 0;
            // Reduce the loop by using a shared memory buffer
            __shared__ int sharedNumMax[LOGREG_ERR_THREADS_X]; // Assumes the number of threads per block is LOGREG_ERR_THREADS_X
            sharedNumMax[threadIdx.x] = 0;
            for (int i = threadIdx.x; i < numOut; i += blockDim.x) {
                sharedNumMax[threadIdx.x] += probs[i * numCases + tx] == maxp;
            }
            __syncthreads();
            // Reduction
            if (threadIdx.x == 0) {
                for (int i = 1; i < LOGREG_ERR_THREADS_X; ++i) {
                    sharedNumMax[0] += sharedNumMax[i];
                }
                numMax = sharedNumMax[0];
            }
            __syncthreads();
            if (threadIdx.x == 0) {
                correctProbs[tx] = 1.0f / float(numMax);
            }
        }
    }
}