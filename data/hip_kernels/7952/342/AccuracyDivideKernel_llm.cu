#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load the value of accuracy into a register
    float acc = *accuracy;
    // Perform the division in register (faster)
    acc /= N;
    // Store the result back to global memory
    *accuracy = acc;
}