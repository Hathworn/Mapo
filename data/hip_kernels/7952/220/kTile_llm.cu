#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Get the global thread ID
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    // Utilize shared memory for optimal memory access
    extern __shared__ float sharedSrc[];
    const uint sharedIndex = threadIdx.x;

    // Each thread loads a single element into shared memory
    if (sharedIndex < srcWidth * srcHeight) {
      sharedSrc[sharedIndex] = src[sharedIndex];
    }
    __syncthreads();

    // Calculate target index and map it to the source matrix
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        tgt[i] = sharedSrc[srcY * srcWidth + srcX];
    }
}