#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel( const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0;

    // Optimized parallel reduction to compute the inner products.
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Use parallel reduction to optimize computation
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (idx < offset) {
            reduction_buffer[idx] += reduction_buffer[idx + offset];
        }
        __syncthreads();
    }

    // Compute gradient.
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}