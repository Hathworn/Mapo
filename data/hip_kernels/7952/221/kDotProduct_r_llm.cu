#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;

    if (eidx < numElements) {
        // Loop unrolling
        for (uint i = eidx; i < numElements; i += gridDim.x * DP_BLOCKSIZE) {
            shmem[threadIdx.x] += a[i] * b[i];
        }
    }

    __syncthreads();

    // Parallel reduction using warp-shuffle
    for (uint offset = DP_BLOCKSIZE / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            shmem[threadIdx.x] += shmem[threadIdx.x + offset];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}