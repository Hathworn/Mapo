#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate the global thread index
    const uint tidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize CURAND state for each thread
    hiprand_init(seed, tidx, 0, &state[tidx]);
}