#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Efficiently divide accuracy by N using parallel reduction
    accuracy[threadIdx.x] /= N; // Thread index used for independent divisions
}