#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    const float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    // Optimize loop with a conditional check outside
    if (thread_id < half_length && thread_id != 0) {
        float2 val = reinterpret_cast<const float2 *>(Ar)[thread_id];
        float ret = val.x * val.x + val.y * val.y;
        Aw[thread_id] = use_power ? ret : sqrtf(ret);
    }

    // Optimize special case handling by using thread synchronization
    __syncthreads();
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}