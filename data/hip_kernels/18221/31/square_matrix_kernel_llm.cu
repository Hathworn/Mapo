#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Calculate global thread index
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if thread is within matrix dimensions
    if (i < num_rows && j < num_cols) {
        float f = feats[i * ldf + j];
        feats_sq[i * lds + j] = f * f; // Compute square
    }
}