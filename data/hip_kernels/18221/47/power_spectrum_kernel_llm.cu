#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    // Calculate unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int half_length = row_length / 2;

    if (idx < half_length) {
        float *Ar = A_in + blockIdx.x * ldi;
        float *Aw = A_out + blockIdx.x * ldo;

        // Special case handled by thread 0
        if (idx == 0) {
            float real = Ar[0];
            float im = Ar[row_length];

            if (use_power) {
                Aw[0] = real * real;
                Aw[half_length] = im * im;
            } else {
                Aw[0] = fabs(real);
                Aw[half_length] = fabs(im);
            }
            return;
        }

        // Calculate power or magnitude for standard case
        float2 val = reinterpret_cast<float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret);
    }
}