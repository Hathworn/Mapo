#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

// Optimized kernel with coalesced memory access
__global__ void vec_add_64(const double* A, const double* B, double* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i]; // Coalesced memory access
    }
}

#ifdef __cplusplus
}
#endif