#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_fma(const float* A, const float* B, float* C, float* D, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Use loop unrolling for better performance
    if (i < N) {
        D[i] = A[i] + B[i] * C[i];
        if (i + blockDim.x < N) D[i + blockDim.x] = A[i + blockDim.x] + B[i + blockDim.x] * C[i + blockDim.x];
        if (i + 2 * blockDim.x < N) D[i + 2 * blockDim.x] = A[i + 2 * blockDim.x] + B[i + 2 * blockDim.x] * C[i + 2 * blockDim.x];
        if (i + 3 * blockDim.x < N) D[i + 3 * blockDim.x] = A[i + 3 * blockDim.x] + B[i + 3 * blockDim.x] * C[i + 3 * blockDim.x];
    }
}