#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_add_64(const double* A, const double* B, double* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Ensure the index is within the bounds
    if (i < N) {
        // Perform the addition operation
        C[i] = A[i] + B[i];
    }
}