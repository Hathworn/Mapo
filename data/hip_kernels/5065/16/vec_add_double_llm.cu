#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif






#ifdef __cplusplus
}
#endif
__global__ void vec_add_double(double *A, double *B, double* C, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for multiple grid iterations

    // Loop over data with stride to ensure all elements are covered
    for (int i = index; i < size; i += stride) {
        C[i] = A[i] + B[i];
    }
}