#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_div(const float* A, const float* B, float* C, int N) {
    // Optimize using a conditional and avoid unnecessary semicolon
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] / B[i]; // Move assignment inside the conditional
    }
}