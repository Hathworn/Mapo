#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_sub(const float* A, const float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Ensure only valid elements are processed
    if (i < N) {
        C[i] = A[i] - B[i];
    }
}