#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_mult(const float* A, const float* B, float* C, int N) {
    // Calculate global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Ensure the thread is within bounds
    if (i < N) {
        // Perform element-wise multiplication
        C[i] = A[i] * B[i];
    }
}