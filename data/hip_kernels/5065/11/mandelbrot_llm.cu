#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif
#ifdef __cplusplus
}
#endif

__global__ void mandelbrot(int* A, const int N, const int largeur, const int hauteur, const int start_hauteur, const int end_hauteur) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    int y = idx / largeur;  // Correct calculation for y coordinate
    int x = idx % largeur;  // Use modulo for x coordinate

    if (y >= start_hauteur && y < end_hauteur) {
        // Initialize constants and variables before the loop
        int cpt = 0;
        float x1 = 0.0f, y1 = 0.0f;
        float a = 4.0f * x / largeur - 2.0f;
        float b = 4.0f * y / hauteur - 2.0f;
        float val = 0.0f;

        // Simplified loop structure, combining initialization with the condition
        do {
            x1 = x1 * x1 - y1 * y1 + a;
            y1 = 2.0f * x1 * y1 + b;
            cpt++;
            val = x1 * x1 + y1 * y1;
        } while (cpt < N && val <= 4.0f);

        A[(y - start_hauteur) * largeur + x] = cpt;  // Corrected index calculation
    }
}