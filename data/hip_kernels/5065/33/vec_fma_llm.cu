#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_fma(const float* A, const float* B, float* C, float* D, int N) {
    // Compute global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread operates within bounds
    if (i < N) {
        // Perform fused multiply-add operation
        D[i] = fmaf(B[i], C[i], A[i]);
    }
}