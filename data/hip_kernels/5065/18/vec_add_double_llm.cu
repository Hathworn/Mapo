#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif

#ifdef __cplusplus
}
#endif
__global__ void vec_add_double(double *A, double *B, double* C, int size)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for better performance
    __shared__ double sA[256];
    __shared__ double sB[256];
    
    if(index < size) {
        // Load data into shared memory
        sA[threadIdx.x] = A[index];
        sB[threadIdx.x] = B[index];
        __syncthreads(); // Ensure all threads have loaded their data

        // Perform the addition
        C[index] = sA[threadIdx.x] + sB[threadIdx.x];
    }
}