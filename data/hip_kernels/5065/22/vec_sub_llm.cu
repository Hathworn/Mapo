#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_sub(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int N) {
    // Use shared memory to store temporary results
    extern __shared__ float cache[];

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Check boundary condition
    if (i < N) {
        // Use registers to minimize global memory access
        float a = A[i];
        float b = B[i];
        
        // Perform subtraction
        C[i] = a - b;
    }
}