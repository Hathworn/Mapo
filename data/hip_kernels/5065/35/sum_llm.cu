#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/





/****** Double precision *****/







#ifdef __cplusplus
}
#endif
__global__ void sum(int* vec1, int* result, int* tmp1, const int count)
{
    // Load and sum elements in shared memory for faster access
    extern __shared__ int shared[];
    int tid = threadIdx.x;
    int global_tid = blockDim.x * blockIdx.x + tid;
    if (global_tid < count)
        shared[tid] = vec1[global_tid];
    else
        shared[tid] = 0; // Initialize padding with zero
    __syncthreads();

    // Use shared memory for parallel reduction
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        if (tid < stride)
        {
            shared[tid] += shared[tid + stride];
        }
        __syncthreads();
    }

    // Only the first thread in each block will add its result to final output
    if (tid == 0)
    {
        atomicAdd(result, shared[0]);
    }
}