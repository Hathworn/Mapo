#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void spoc_max(const double* input, double* output, const int size)
{
    // Use shared memory to store max values
    extern __shared__ double shared_max[];

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Initialize shared memory
    shared_max[threadIdx.x] = (i < size) ? fabs(input[i]) : 0.0;
    __syncthreads();

    // Reduce to find the maximum
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (threadIdx.x % (2 * s) == 0 && (threadIdx.x + s) < blockDim.x) {
            if (shared_max[threadIdx.x] < shared_max[threadIdx.x + s]) {
                shared_max[threadIdx.x] = shared_max[threadIdx.x + s];
            }
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        atomicMax((unsigned long long int*)&output[0], __double_as_longlong(shared_max[0]));
    }
}