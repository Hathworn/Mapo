#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

#ifdef __cplusplus
}
#endif

__global__ void init(double* A, double* V, double* VN, int size, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= size) return;

    double s = 0.0;
    int stride = n; // Pre-calculate stride to improve readability
    int index = i * stride; // Pre-compute base index for row
    for (int j = 0; j < n; j++) {
        s += A[index + j] * V[j]; // Use pre-computed index
    }
    VN[i] = s;
}