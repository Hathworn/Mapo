#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

// Optimized kernel function for vector division
__global__ void vec_div(const float* A, const float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure index is within bounds and calculate division
    if (i < N) {
        C[i] = A[i] / B[i];
    }
}