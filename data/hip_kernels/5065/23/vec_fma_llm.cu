#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_fma(const float* A, const float* B, float* C, float* D, int N) {
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check bounds and perform operation if within range
    if (i < N) {
        D[i] = A[i] + B[i] * C[i];
    }
}