#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif


#ifdef __cplusplus
}
#endif
__global__ void kernel_compute(int* trainingSet, int* data, int* res, int setSize, int dataSize){
    int computeId = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure computeId is within setSize bounds
    if(computeId < setSize){
        int diff = 0;

        // Use register variable for loop to avoid repeated computations
        int offset = computeId * 784;

        for(int i = 0; i < dataSize; i++){
            int toAdd = data[i] - trainingSet[offset + i];
            diff += toAdd * toAdd;
        }

        // Write result directly after computation
        res[computeId] = diff;
    }
}