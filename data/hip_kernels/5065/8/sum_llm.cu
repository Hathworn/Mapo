#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/******************************************************************************
* Mathias Bourgoin, Université Pierre et Marie Curie (2011)
*
* Mathias.Bourgoin@gmail.com
*
* This software is a computer program whose purpose is to allow
* GPU programming with the OCaml language.
*
* This software is governed by the CeCILL-B license under French law and
* abiding by the rules of distribution of free software.  You can  use,
* modify and/ or redistribute the software under the terms of the CeCILL-B
* license as circulated by CEA, CNRS and INRIA at the following URL
* "http://www.cecill.info".
*
* As a counterpart to the access to the source code and  rights to copy,
* modify and redistribute granted by the license, users are provided only
* with a limited warranty  and the software's author,  the holder of the
* economic rights,  and the successive licensors  have only  limited
* liability.
*
* In this respect, the user's attention is drawn to the risks associated
* with loading,  using,  modifying and/or developing or reproducing the
* software by the user in light of its specific status of free software,
* that may mean  that it is complicated to manipulate,  and  that  also
* therefore means  that it is reserved for developers  and  experienced
* professionals having in-depth computer knowledge. Users are therefore
* encouraged to load and test the software's suitability as regards their
* requirements in conditions enabling the security of their systems and/or
* data to be ensured and,  more generally, to use and operate it in the
* same conditions as regards security.
*
* The fact that you are presently reading this means that you have had
* knowledge of the CeCILL-B license and that you accept its terms.
*******************************************************************************/
#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/





/****** Double precision *****/







#ifdef __cplusplus
}
#endif
__global__ void sum(int * vec1, int * result, int* tmp1, const int count)
{
    // Calculate global thread ID
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < count / 2)
    {
        // Perform initial sum and write to shared memory
        tmp1[tid] = vec1[tid] + vec1[tid + count / 2];
    }
    __syncthreads(); // Ensure all writes to tmp1 are complete

    // Parallel reduction within the block using shared memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride && tid + stride < count / 2)
        {
            tmp1[tid] += tmp1[tid + stride];
        }
        __syncthreads();
    }

    // Write the result of the block's reduction
    if (tid == 0)
    {
        atomicAdd(result, tmp1[0]);
    }
}