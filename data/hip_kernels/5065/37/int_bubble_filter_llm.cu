#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

__global__ void int_bubble_filter(int* input, const int* vec1, int* output, const int count) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = 2 * tid;
    if (offset < count) {
        // Initialize the output with vec1
        output[offset] = vec1[offset];
        output[offset + 1] = vec1[offset + 1];

        bool k;
        for (int n = 0; n < count; ++n) {
            k = n % 2;
            int i = offset + k;
            if (i + 1 < count) {
                if (!input[i] && input[i + 1]) {
                    // Swap elements
                    input[i] = 1;
                    input[i + 1] = 0;
                    output[i] = output[i + 1];
                    output[i + 1] = 0;
                }
                else {
                    // Clear output if needed
                    if (!input[i]) output[i] = 0;
                    if (!input[i + 1]) output[i + 1] = 0;
                }
            }
            __syncthreads();  // Synchronize threads
        }
    }
}

#ifdef __cplusplus
}
#endif