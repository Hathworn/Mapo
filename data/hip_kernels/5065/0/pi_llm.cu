#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif

struct point {
    float x;
    float y;
};

struct point2 {
    double x;
    double y;
};

#ifdef __cplusplus
}
#endif

__global__ void pi(const struct point* A, int* res, const int nbPoint, const float ray) {
    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check bound; original had extra constant 32, corrected here for better bounds management
    if (idx < nbPoint) {
        // Precompute ray squared to avoid recomputing in loop
        float ray_squared = ray * ray;

        // Iterate and store results using block size of stride
        #pragma unroll 16
        for (int j = 0; j < 32 && idx + j < nbPoint; j++) {
            int i = idx + blockDim.x * j;
            res[i] = (A[i].x * A[i].x + A[i].y * A[i].y <= ray_squared);
        }
    }
}