#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif



#ifdef __cplusplus
}
#endif
__global__ void game(int* A, const int N, const int largeur, const int hauteur) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= N) return;  // Avoid accessing out of bounds

    int y = idx / largeur;
    int x = idx % largeur;

    int me = A[idx];
    int north = (y > 0) ? A[idx - largeur] : 0;
    int south = (y < hauteur - 1) ? A[idx + largeur] : 0;
    int east = (x < largeur - 1) ? A[idx + 1] : 0;
    int west = (x > 0) ? A[idx - 1] : 0;
    int northEast = (y > 0 && x < largeur - 1) ? A[idx - largeur + 1] : 0;
    int northWest = (y > 0 && x > 0) ? A[idx - largeur - 1] : 0;
    int southEast = (y < hauteur - 1 && x < largeur - 1) ? A[idx + largeur + 1] : 0;
    int southWest = (y < hauteur - 1 && x > 0) ? A[idx + largeur - 1] : 0;

    // Calculate neighboring cells' sum
    int res = north + south + east + west + northEast + northWest + southEast + southWest;

    // Update the current cell's state
    if ((me == 1 && (res < 2 || res > 3)) || (me == 0 && res == 3))
        A[idx] = 0;
    else if (me == 0 && res == 3)
        A[idx] = 1;
}