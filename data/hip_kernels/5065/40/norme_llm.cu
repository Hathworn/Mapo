#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

#ifdef __cplusplus
}
#endif

__global__ void norme(const double* VN, double * V, double* v_norme, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure that the index is within bounds before processing
    if (i < size) {
        // Directly compute the difference and update both arrays
        double value_VN = VN[i];
        double value_V = V[i];
        v_norme[i] = fabs(value_VN - value_V);
        V[i] = value_VN;
    }
}