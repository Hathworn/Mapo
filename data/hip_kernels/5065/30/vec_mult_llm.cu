#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_mult(const float* A, const float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        // Perform element-wise multiplication
        C[i] = A[i] * B[i];
    }
}