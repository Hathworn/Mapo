#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

__global__ void int_bubble_filter(int* input, const int* vec1, int* output, const int count)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Improved boundary check to reduce warp divergence
    if (tid >= count / 2) return;

    // Optimized initialization using shared memory for reduced global memory access
    extern __shared__ int shared[];
    shared[threadIdx.x * 2] = vec1[tid * 2];
    shared[threadIdx.x * 2 + 1] = vec1[tid * 2 + 1];
    __syncthreads();

    int k = 1;
    for (int n = 0; n < count * 2; n++)
    {
        k = 1 - k;  // Toggle k between 0 and 1

        int i = (tid * 2) + k;
        if (i + 1 < count)
        {
            if (!input[i] && input[i + 1])
            {
                input[i] = 1;
                input[i + 1] = 0;
                output[i] = shared[threadIdx.x * 2 + 1];  // Use shared memory
                shared[threadIdx.x * 2 + 1] = 0;
            }
            else
            {
                if (!input[i])
                    output[i] = 0;
                if (!input[i + 1])
                    output[i + 1] = 0;
            }
        }
        __syncthreads();
    }

    // Move final results from shared memory back to output
    output[tid * 2] = shared[threadIdx.x * 2];
    output[tid * 2 + 1] = shared[threadIdx.x * 2 + 1];
}

#ifdef __cplusplus
}
#endif