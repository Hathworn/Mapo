#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_add(const float* A, const float* B, float* C, int N) {
    // Calculate global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if within bounds, then perform the addition
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}