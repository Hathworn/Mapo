#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif



#ifdef __cplusplus
}
#endif
__global__ void mandelbrot(int* A, const int N, const int largeur, const int hauteur) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int total_pixels = largeur * hauteur;

    // Ensure idx does not exceed total number of pixels
    if (idx < total_pixels) {
        int y = idx / largeur; // Correct computation of y coordinate
        int x = idx % largeur; // Correct computation of x coordinate

        int cpt = 0;
        float x1 = 0.;
        float y1 = 0.;

        float a = 4.0f * x / largeur - 2.0f;
        float b = 4.0f * y / hauteur - 2.0f;

        float val = 0.0f;

        // Unroll the loop for potential performance improvement
        #pragma unroll
        while (cpt < N && val <= 4.0f) {
            cpt++;
            float x2 = x1 * x1 - y1 * y1 + a;
            y1 = 2.0f * x1 * y1 + b;
            x1 = x2;
            val = x1 * x1 + y1 * y1;
        }

        A[y * largeur + x] = cpt; // Correct storage location in output array
    }
}