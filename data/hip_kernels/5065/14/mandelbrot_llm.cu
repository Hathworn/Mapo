#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif
#ifdef __cplusplus
}
#endif
__global__ void mandelbrot(int* A, const int N, const int largeur, const int hauteur) {
    // Calculate global thread indices
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds
    if (y < hauteur && x < largeur) {
        int cpt = 0;
        float x1 = 0.f, y1 = 0.f, x2 = 0.f, y2 = 0.f;
        float a = 4.f * x / largeur - 2.f;
        float b = 4.f * y / hauteur - 2.f;
        
        // Calculate squared magnitude in advance
        float val = x1 * x1 + y1 * y1;
        
        // Perform iteration
        while (cpt < N && val <= 4.f) {
            cpt++;
            x2 = x1 * x1 - y1 * y1 + a;
            y2 = 2.f * x1 * y1 + b;
            x1 = x2;
            y1 = y2;
            val = x1 * x1 + y1 * y1;  // Update squared magnitude
        }
        
        // Store the result
        A[y * largeur + x] = cpt;  // Correct indexing for 2D array in flat memory
    }
}