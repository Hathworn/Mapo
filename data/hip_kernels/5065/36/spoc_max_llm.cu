#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void spoc_max(const double* input, double* output, const int size)
{
    // Use shared memory to reduce global memory accesses
    extern __shared__ double sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    // Load data into shared memory
    if (i < size) {
        sdata[tid] = fabs(input[i]);
    } else {
        sdata[tid] = 0.0;  // Set to 0 if out of bounds
    }
    __syncthreads();

    // Perform parallel reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = fmax(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    // First thread writes the result to the output
    if (tid == 0) {
        atomicMax((unsigned long long int*)output, __double_as_longlong(sdata[0]));
    }
}