#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_sub(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    if (i < N) {  // Ensure thread is within array bounds
        C[i] = A[i] - B[i];  // Perform the subtraction and store result
    }
}