#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif






#ifdef __cplusplus
}
#endif
__global__ void vec_add(float *A, float *B, float* C, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for performance optimization
    extern __shared__ float shared[];

    if (index < size) {
        // Load the vectors from global memory to shared memory
        shared[threadIdx.x] = A[index];
        shared[threadIdx.x + blockDim.x] = B[index];

        // Synchronize to ensure all threads have loaded their data
        __syncthreads();

        // Perform the addition and store the result back to global memory
        C[index] = shared[threadIdx.x] + shared[threadIdx.x + blockDim.x];
    }
}