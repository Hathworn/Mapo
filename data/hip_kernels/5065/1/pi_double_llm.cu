#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif

struct point {
    float x;
    float y;
};

struct point2 {
    double x;
    double y;
};

#ifdef __cplusplus
}
#endif

__global__ void pi_double(const struct point2* A, int* res, const int nbPoint, const float ray) {
    // Calculate global thread index
    const int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Unrolling factor
    const int unrollFactor = 32;

    // Calculate number of iterations per thread
    const int iterations = (nbPoint + blockDim.x * unrollFactor - 1) / (blockDim.x * unrollFactor);

    #pragma unroll
    for (int i = 0; i < iterations; i++) {
        int idx = globalIdx + blockDim.x * i;
        
        // Boundary check
        if (idx < nbPoint) {
            double distanceSquared = A[idx].x * A[idx].x + A[idx].y * A[idx].y;
            res[idx] = (distanceSquared <= (double)ray);
        }
    }
}