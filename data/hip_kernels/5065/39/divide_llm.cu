#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

#ifdef __cplusplus
}
#endif

__global__ void divide(double* VN, const double* max, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size) { // Combine condition check to eliminate redundant else
        VN[i] = VN[i] / max[0];
    }
}