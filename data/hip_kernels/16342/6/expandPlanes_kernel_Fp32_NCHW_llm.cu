#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void expandPlanes_kernel_Fp32_NCHW(float* output, const uint64_t* masks, const float* values, int n) {
    // Block size of 256, same mask/val for 64 consecutive threads.
    constexpr int kNumShmemElements = 256 / 64;

    __shared__ uint64_t shMasks[kNumShmemElements];
    __shared__ float shVals[kNumShmemElements];

    int index = threadIdx.x + blockDim.x * blockIdx.x;

    int planeIndex = index >> 6;
    if (planeIndex >= n) return;

    // Load inputs to shared memory efficiently.
    if (threadIdx.x < kNumShmemElements) {
        shMasks[threadIdx.x] = masks[planeIndex + threadIdx.x];
        shVals[threadIdx.x] = values[planeIndex + threadIdx.x];
    }
    __syncthreads();

    // Efficient mask and computation.
    uint64_t mask = shMasks[planeIndex];  // Use correct index for shared memory
    int sqIndex = index & 0x3F;
    float op = 0;

    bool set = !!(mask & (1ull << sqIndex));
    if (set) {
        op = shVals[planeIndex];  // Use correct index for shared memory
    }
    output[index] = op;
}