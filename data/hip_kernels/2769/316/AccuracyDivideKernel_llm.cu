#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use the first thread of the first block to perform the division
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}