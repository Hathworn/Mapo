#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rearrangePopulationWithRange(float *gene, float *fit, int *range)
{
    const int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (range[0] > range[1]) return;   // Return if range is invalid

    int totalElements = range[1] - range[0] + 1;
    int nHalf = totalElements / 2;
    if (idx >= nHalf) return;  // Ensure threads beyond the valid range do not process
   
    int i = range[0] + idx;
    int j = range[1] - idx;

    if (fit[i] < fit[j]) {
        // Use shared memory for faster access within a block
        __shared__ float tempGene[6 * 256];  // Assuming blockDim.x is at most 256
        __shared__ float tempFit[256];

        for (int k = 0; k < 6; k++) {
            tempGene[threadIdx.x * 6 + k] = gene[i * 6 + k];
        }
        tempFit[threadIdx.x] = fit[i];

        __syncthreads();

        for (int k = 0; k < 6; k++) {
            gene[i * 6 + k] = gene[j * 6 + k];
            gene[j * 6 + k] = tempGene[threadIdx.x * 6 + k];
        }
        fit[i] = fit[j];
        fit[j] = tempFit[threadIdx.x];
    }
}