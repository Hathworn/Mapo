#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rearrangePopulation(float *gene, float *fit, int* metaData)
{
    const int idx = threadIdx.x + blockDim.x * blockIdx.x;
    const int nGene = metaData[1];
    const int nHalf = nGene / 2;
    if (idx >= nHalf) return;  // Bounds checking

    const int j = nGene - 1 - idx;

    if (fit[idx] < fit[j]) {
        // Swapping genes
        for (int k = 0; k < 6; k++) {
            float t = gene[idx * 6 + k];
            gene[idx * 6 + k] = gene[j * 6 + k];
            gene[j * 6 + k] = t;
        }
        // Swapping fit values
        float tFit = fit[idx];
        fit[idx] = fit[j];
        fit[j] = tFit;
    }
}