#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rearrangePopulation(float *gene, float *fit, int* metaData)
{
    const int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int nGene = metaData[1];
    int nHalf = nGene / 2;
    
    if (idx >= nHalf) 
        return; // Terminate threads that are out of range

    int j = nGene - 1 - idx;

    if (fit[idx] < fit[j]) {
        // Swap elements using temporary variables
        for (int k = 0; k < 6; k++) {
            float t = gene[idx * 6 + k];
            gene[idx * 6 + k] = gene[j * 6 + k];
            gene[j * 6 + k] = t;
        }

        float tFit = fit[idx];
        fit[idx] = fit[j];
        fit[j] = tFit;
    }
}