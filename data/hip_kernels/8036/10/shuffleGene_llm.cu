#include "hip/hip_runtime.h"
```c++
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shuffleGene(float *gene, float *fit, float *rSeed, int* metaData) {
    const int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int nGene = metaData[1];
    int nHalf = nGene / 2;
    if (idx >= nHalf) return;  // Corrected condition to include idx == nHalf

    int Offset = int(nHalf / 5.3);
    int j = nHalf + (idx + Offset) % nHalf;

    // Eliminate repeated memory accesses by introducing temporary variables
    float tmpGene[6];
    for (int k = 0; k < 6; k++) {
        tmpGene[k] = gene[j * 6 + k];
    }

    for (int k = 0; k < 6; k++) {
        float t = gene[idx * 6 + k];
        gene[idx * 6 + k] = tmpGene[k];
        gene[j * 6 + k] = t;
    }
   
    // Swap fitness values once outside loop
    float tFit = fit[idx];
    fit[idx] = fit[j];
    fit[j] = tFit;
}