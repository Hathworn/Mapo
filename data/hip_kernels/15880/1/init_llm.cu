#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(int order, double * C)
{
    // Calculate global thread ID for 2D grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensuring within bounds and initializing C matrix elements
    if (i < order && j < order) {
        C[i * order + j] = 0.0;
    }
}