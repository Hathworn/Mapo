#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int* c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < N) // Ensure within bounds, assume N is the total number of elements
    {
        c[i] = a[i] + b[i];
    }
}