#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vector_addition(int *a, int *b, int *c, int n) {
    // Calculate global index for the element
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread index is within range
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}