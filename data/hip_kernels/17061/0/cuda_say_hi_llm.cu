#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_say_hi() {
    // Minimize thread divergence by ensuring only one thread executes the print statement
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Hi, CUDA!\n");
    }
}