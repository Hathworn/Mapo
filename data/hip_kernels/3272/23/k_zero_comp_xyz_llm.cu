#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void k_zero_comp_xyz(float *data, uint n, uint stride)
{
    // Calculate the global thread ID
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the starting index for data pointer based on 'p'
    uint baseIndex = blockIdx.y * stride;

    // Ensure the thread index is within bounds
    if (i < n) {
        // Access the data array with precomputed index
        data[baseIndex + i] = 0.f;
    }
}