#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void k_reorder_send_buf_total(int nr_prts, int nr_total_blocks, uint* d_bidx, uint* d_sums, float4* d_xi4, float4* d_pxi4, float4* d_xchg_xi4, float4* d_xchg_pxi4) 
{
    int i = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

    // Early exit if thread index exceeds number of particles
    if (i >= nr_prts) return;

    // Only proceed if particle is out-of-bounds
    if (d_bidx[i] == CUDA_BND_S_OOB) 
    {   
        // Use shared memory optimization
        __shared__ float4 shared_xi4[THREADS_PER_BLOCK];
        __shared__ float4 shared_pxi4[THREADS_PER_BLOCK];

        // Load data into shared memory
        shared_xi4[threadIdx.x] = d_xi4[i];
        shared_pxi4[threadIdx.x] = d_pxi4[i];

        // Ensure all threads have loaded their data
        __syncthreads();

        // Write data from shared to global memory
        int j = d_sums[i];
        d_xchg_xi4[j] = shared_xi4[threadIdx.x];
        d_xchg_pxi4[j] = shared_pxi4[threadIdx.x];
    }
}
```
