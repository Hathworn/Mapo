#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void mprts_update_offsets(int nr_total_blocks, uint* d_off, uint* d_spine_sums)
{
  // Calculate global thread index
  int bid = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

  // Ensure to stay within bounds
  if (bid < nr_total_blocks) {
    // Read and update offsets
    d_off[bid] = d_spine_sums[bid * CUDA_BND_STRIDE];
  }
}