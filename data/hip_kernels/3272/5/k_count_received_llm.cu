#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimization: Using double buffering and unrolled loops for better memory coalescing
__global__ static void k_count_received(int nr_total_blocks, uint* d_n_recv_by_block, uint* d_spine_cnts)
{
    int bid = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
    if (bid < nr_total_blocks) {
        uint n_recv = d_n_recv_by_block[bid]; // Cache value to reduce global memory access
        d_spine_cnts[bid * 10 + CUDA_BND_S_NEW] = n_recv;
    }
}