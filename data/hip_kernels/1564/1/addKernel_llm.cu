#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel using block and thread indices to handle larger arrays
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Get global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Perform addition for this thread's designated index
    c[i] = a[i] + b[i];
}