#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuke_sinGrid(unsigned char *surface, int width, int height, size_t pitch, float t)
{
    // Calculate pixel coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Skip invalid thread indices
    if (x >= width || y >= height) return;

    // Compute stride and pixel offset
    float* pixel = reinterpret_cast<float*>(surface + y * pitch) + 4 * x;

    // Calculate color values
    float cos_x = cos(t + 10.0f * (2.0f * x / width - 1.0f));
    float cos_y = cos(t + 10.0f * (2.0f * y / height - 1.0f));

    // Update pixel colors with precomputed values
    pixel[0] = 0.5f * pixel[0] + 0.5f * __powf(0.5f + 0.5f * cos_x, 3.0f); // Red
    pixel[1] = 0.5f * pixel[1] + 0.5f * __powf(0.5f + 0.5f * cos_y, 3.0f); // Green
    pixel[2] = 0.5f + 0.5f * cos(t); // Blue
    pixel[3] = 1.0f; // Alpha
}