#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define PI 3.1415926536f

texture<float, 2, hipReadModeElementType> texRef;

//=================================
// write to texture;
//=================================
enum colors
{
RED, GREEN, BLUE, ALPHA
};

__global__ void cuke_gradient(unsigned char *surface, int width, int height, size_t pitch, float t)
{
    // Calculate unique thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if thread corresponds to a valid pixel
    if (x >= width || y >= height) return;

    // Efficiently calculate pixel pointer
    float* pixel = reinterpret_cast<float*>(surface + y * pitch) + 4 * x;

    // Write pixel data
    pixel[RED]   = x / 640.0f;    // Normalize x coordinate
    pixel[GREEN] = y / 480.0f;    // Normalize y coordinate
    pixel[BLUE]  = 0.0f;          // Constant blue value
    pixel[ALPHA] = 1.0f;          // Constant alpha value
}