#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

// Optimized Kernel Average with Depth
__global__ void AVERAGE_DEPTH_1D(int envSizeX, int envSizeY, float* envData, int depth) {
    int tidX = blockIdx.x * blockDim.x + threadIdx.x;
    int tidY = blockIdx.y * blockDim.y + threadIdx.y;

    float moyenne = 0;
    int nbNombre = 0;

    // Check if thread is within bounds
    if (tidX < envSizeX && tidY < envSizeY) {
        // Iterate over neighborhood region
        for (int l = -depth; l <= depth; l++) {
            int lIndex = (tidX + l + envSizeX) % envSizeX; // Use modulo for circular boundary

            for (int k = -depth; k <= depth; k++) {
                int kIndex = (tidY + k + envSizeY) % envSizeY; // Use modulo for circular boundary

                float val = envData[envSizeX * lIndex + kIndex]; // Access data directly
                if (val != -1) {
                    moyenne += val;
                    nbNombre++;
                }
            }
        }

        // Calculate average if count is non-zero
        if (nbNombre != 0) {
            envData[envSizeX * tidX + tidY] = moyenne / nbNombre;
        }
    }
    __syncthreads();
}