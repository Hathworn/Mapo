#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv2genericrev(float *input, float *kernel, float *output, int input_n, int input_h, int input_w, int kernel_n, int kernel_h, int kernel_w, float alpha, int stride_h, int stride_w)
{
    // output dimensions
    int output_h = input_h - (kernel_h - 1) * stride_h;
    int output_w = input_w - (kernel_w - 1) * stride_w;

    // thread indices and output offset
    int kk = blockIdx.x;
    int ii = blockIdx.y;
    int batch = threadIdx.z;
    int kid = threadIdx.x;
    int nkids = blockDim.x;
    int tid = kid + batch * blockDim.x;
    int nthreads = blockDim.x * blockDim.z;

    // Offset output for current (batch, kk, ii)
    output = output + (kk * input_n + ii) * output_h * output_w;

    // Shared memory for output
    __shared__ float shared_output[CUDA_SHARED_MEM_SIZE];
    float *output_s = shared_output + tid * output_w * output_h;

    int yy = threadIdx.y;
    float *output_p = output_s + yy * output_w;

    for(int xx = 0; xx < output_w; xx++) {
        float *input_p = input + (ii + batch*input_n) * input_h * input_w + yy * stride_h * input_w + xx * stride_w;
        float *kernel_p = kernel + (kk + batch * kernel_n) * kernel_w * kernel_h;
        float sum = 0;

        // Use loop unrolling to enhance performance
        #pragma unroll
        for(int ky = 0; ky < kernel_h; ky++) {
            for(int kx = kid; kx < kernel_w; kx += nkids) {
                sum += input_p[kx] * kernel_p[kx];
            }
            input_p += input_w;
            kernel_p += kernel_w;
        }
        *(output_p++) = sum;
    }
    __syncthreads();

    // Reduce and write back
    if (yy == 0) {
        for (int k = 1; k < nthreads; k++) {
            for (int i = tid; i < output_w * output_h; i += nthreads) {
                shared_output[i] += shared_output[k * output_h * output_w + i];
            }
        }
        __syncthreads();

        for (int i = tid; i < output_w * output_h; i += nthreads) {
            output[i] += alpha * shared_output[i];
        }
    }
}