#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Use local variable for tidx to minimize repeated calculation
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    // Initialize hiprand with optimized sequence number calculation
    hiprand_init(seed, tidx, 0, &state[tidx]);
}