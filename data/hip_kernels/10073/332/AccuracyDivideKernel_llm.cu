#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use atomic operation to prevent race condition
    atomicExch(accuracy, *accuracy / N);
}