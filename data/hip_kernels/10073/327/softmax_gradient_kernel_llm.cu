#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // Unroll loop to improve performance.
    for (int i = idx; i < dim; i += blockDim.x * 4) {
        tmp += (i < dim) ? (dY[i] * Y[i]) : 0.0f;
        if (i + blockDim.x < dim) tmp += dY[i + blockDim.x] * Y[i + blockDim.x];
        if (i + 2 * blockDim.x < dim) tmp += dY[i + 2 * blockDim.x] * Y[i + 2 * blockDim.x];
        if (i + 3 * blockDim.x < dim) tmp += dY[i + 3 * blockDim.x] * Y[i + 3 * blockDim.x];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Reduce to single sum using warp-level reduction.
    if (idx < warpSize) { 
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            tmp += __shfl_down(tmp, offset);
        }
    }
    if (idx == 0) {
        reduction_buffer[0] = tmp;
    }
    __syncthreads();

    // Compute gradient using the reduced sum.
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}