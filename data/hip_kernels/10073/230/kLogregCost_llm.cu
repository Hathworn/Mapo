#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Optimize: Utilize shared memory for numMax calculation.
        extern __shared__ int sharedNumMax[];

        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            for (int i = threadIdx.x; i < numOut; i += blockDim.x) {
                // Use atomic operation to safely update shared memory.
                atomicAdd(&sharedNumMax[threadIdx.x], probs[i * numCases + tx] == maxp);
            }
            __syncthreads();

            // Reduce shared memory results.
            if (threadIdx.x == 0) {
                for (int i = 0; i < blockDim.x; i++) {
                    numMax += sharedNumMax[i];
                }
                correctProbs[tx] = 1.0f / float(numMax);
            }
        }
    }
}