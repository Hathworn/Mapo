#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Optimize thread indexing by computing once and removing redundant division
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    const uint numEls = tgtWidth * tgtHeight;
    const uint srcArea = srcWidth * srcHeight;

    for (uint i = idx; i < numEls; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        // Prevent redundant modulus operation by using pre-computed srcArea
        const uint srcY = y < srcHeight ? y : y % srcHeight;
        const uint srcX = x < srcWidth ? x : x % srcWidth;
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}