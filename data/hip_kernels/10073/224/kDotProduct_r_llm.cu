#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint tid = threadIdx.x;
    uint eidx = DP_BLOCKSIZE * blockIdx.x + tid;
    
    // Initialize shared memory for this thread
    float sum = 0.0f;

    // Each thread handles multiple elements for better memory coalescing
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        sum += a[eidx] * b[eidx];
    }
    
    // Store the sum in shared memory
    shmem[tid] = sum;

    // Reduction using shared memory
    __syncthreads();
    // Optimized reduction loop for power of two block sizes
    for (uint stride = DP_BLOCKSIZE / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            shmem[tid] += shmem[tid + stride];
        }
        __syncthreads();
    }

    // Write the result to the target
    if (tid == 0) {
        target[blockIdx.x] = shmem[0];
    }
}