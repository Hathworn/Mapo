#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define PI 3.1415926536f

__global__ void KGaborFilter1(double* filter, double* Vr, int width, int height, double ss, double r0, double sr0, double stheta0)
{
    // Calculate global thread indices and check bounds
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= width || j >= height) return;

    // Determine x and y positions relative to the center
    double x = (i > width / 2) ? width - i : i;
    double y = (j > height / 2) ? height - j : j;

    // Compute radius with offset to prevent division by zero
    const double Eps = 1E-6;
    double r = sqrt(x * x + y * y) + Eps;

    // Compute angle
    double theta = (x > 0) ? atan2(y, x) : PI / 2;

    // Calculate filter value
    double ff = exp(cos(2 * theta) / (4 * pow(stheta0, 2))) *
                exp(-0.5 * pow(log(r / r0), 2) / log(1 + pow(sr0, 2))) *
                pow(r0 / r, 3) * 4 * pow(ss * r, 3);

    // Assign the calculated value to filter and Vr arrays
    filter[i + j * width] = ff;
    Vr[i + j * width] = (i > 0 || j > 0) ? ff / (4 * pow(ss * r, 3)) : 0;
}