#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define PI 3.1415926536f

int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;

__global__ void FillTex(void *surface, int width, int height, size_t pitch, double* src, int Mask)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread does not access out-of-bounds memory
    if (x >= width || y >= height) return;

    double w = src[x + width * y];
    unsigned char *pixel1 = (unsigned char *)((char*)surface + y * pitch) + 4 * x;

    // Clamping w value within range [0, 253]
    w = min(max(w, 0.0), 253.0);

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        if (Mask & (1 << i)) {
            pixel1[i] = (unsigned char)w;
        }
    }
}