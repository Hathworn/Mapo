#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define PI 3.1415926536f

int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;

__global__ void KtexFillRect(void* surface, double* tb, int width, int height, size_t pitch, float2* Pts, int k, float th) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Early exit if out of bounds
    if (x >= width || y >= height) return;

    unsigned char *pixel1 = (unsigned char *)((char*)surface + y * pitch) + 4 * x;

    // Check if point is inside the polygon
    bool insidePolygon = ((Pts[1].y - Pts[0].y) * (x - Pts[0].x) - (y - Pts[0].y) * (Pts[1].x - Pts[0].x) >= 0) &&
                         ((Pts[2].y - Pts[1].y) * (x - Pts[1].x) - (y - Pts[1].y) * (Pts[2].x - Pts[1].x) >= 0) &&
                         ((Pts[3].y - Pts[2].y) * (x - Pts[2].x) - (y - Pts[2].y) * (Pts[3].x - Pts[2].x) >= 0) &&
                         ((Pts[0].y - Pts[3].y) * (x - Pts[3].x) - (y - Pts[3].y) * (Pts[0].x - Pts[3].x) >= 0);

    if (insidePolygon && (pixel1[k] >= th)) {
        tb[x + width * y] = 1;
    }
}