#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define PI 3.1415926536f

int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;

__global__ void  KparamAR(double* a, double* b, double* c, double ss, double dtAR, int width, int height) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check boundaries at the start to avoid unnecessary computation
    if (i >= width || j >= height) return;

    // Compute symmetric indices efficiently
    int x = (i > width / 2) ? width - i : i;
    int y = (j > height / 2) ? height - j : j;

    // Use pre-calculated value for similar operations
    double r_squared = (double)x*x + (double)y*y;
    double r = sqrt(r_squared) + EPS; // Precalculate sqrt value

    // Remove redundant calculations
    double dtAR_ss_r = dtAR * ss * r;
    a[i + j * width] = 2 - dtAR_ss_r * 2 - dtAR_ss_r * dtAR_ss_r;
    b[i + j * width] = -1 + dtAR_ss_r * 2;
    c[i + j * width] = 1;
}