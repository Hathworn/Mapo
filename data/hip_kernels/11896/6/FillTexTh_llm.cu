#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define PI 3.1415926536f

int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;

__global__ void FillTexTh(void *surface, int width, int height, size_t pitch, double* src, int Mask, int th, int pixValue)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    unsigned char *pixel1 = (unsigned char *)((char*)surface + y * pitch) + 4 * x;

    double w = src[x + width*y];
    w = fminf(fmaxf(w, 0.0), 253.0); // Optimize w clamping using intrinsic functions

    if (pixel1[3] >= th) {
        for (int i = 0; i < 3; i++) {
            if (Mask & (1 << i)) pixel1[i] = static_cast<unsigned char>(w); // Simplified conversion to unsigned char
        }
    } else {
        for (int i = 0; i < 3; i++) {
            if (Mask & (1 << i)) pixel1[i] = static_cast<unsigned char>(pixValue >> (i * 8)); // Simplified conversion to unsigned char
        }
    }
}