#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define PI 3.1415926536f

int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;

// Optimized Kernel function
__global__ void Kernel_Expansion1(double *tb1, double *tb2, int width, int height, double Dx, double x0, double Dy, double y0)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    // Precompute terms that are used more than once
    double invDx = 1.0 / Dx;
    double invDy = 1.0 / Dy;

    double xt = (x - x0) * invDx + x0;
    double yt = (y - y0) * invDy + y0;

    int x1 = static_cast<int>(xt) % width;
    int y1 = static_cast<int>(yt) % height;

    int xp1 = (x1 + 1) % width;
    int yp1 = (y1 + 1) % height;

    // Load values with less index calculations
    double dx = xt - floor(xt);
    double dy = yt - floor(yt);

    double z1 = tb1[width * y1 + x1];
    double z2 = tb1[width * yp1 + x1];
    double z3 = tb1[width * yp1 + xp1];
    double z4 = tb1[width * y1 + xp1];

    // Reduce redundant calculations
    double zp = z1 + dy * (z2 - z1);
    double zq = z4 + dy * (z3 - z4);
    double ZR = zp + dx * (zq - zp);

    tb2[width * y + x] = ZR;
}