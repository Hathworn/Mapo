#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "Kernel_Expansion1_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    double *tb1 = NULL;
hipMalloc(&tb1, XSIZE*YSIZE);
double *tb2 = NULL;
hipMalloc(&tb2, XSIZE*YSIZE);
int width = XSIZE;
int height = YSIZE;
double Dx = 1;
double x0 = 1;
double Dy = 1;
double y0 = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
Kernel_Expansion1<<<gridBlock, threadBlock>>>(tb1,tb2,width,height,Dx,x0,Dy,y0);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
Kernel_Expansion1<<<gridBlock, threadBlock>>>(tb1,tb2,width,height,Dx,x0,Dy,y0);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
Kernel_Expansion1<<<gridBlock, threadBlock>>>(tb1,tb2,width,height,Dx,x0,Dy,y0);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}