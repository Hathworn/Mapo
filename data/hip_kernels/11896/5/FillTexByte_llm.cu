#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define PI 3.1415926536f

int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;

// Optimized FillTexByte kernel
__global__ void FillTexByte(void *surface, int width, int height, size_t pitch, char* src, int Mask)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Return if the thread is out of bounds
    if (x >= width || y >= height) return;

    // Calculate linear index for source
    char w = src[x + width * y];

    // Calculate pixel pointer address in surface
    unsigned char *pixel1 = (unsigned char *)((char *)surface + y * pitch) + 4 * x;

    // Apply mask and update pixel value
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        if (Mask & (1 << i)) {
            pixel1[i] = w;
        }
    }
}