#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define PI 3.1415926536f

int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;

__global__ void Kernel_CartToPol1(double *tb1, double *tb2, int width, int height)
{
    // Calculate global thread coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary check
    if (x >= width || y >= height) return;

    // Pre-compute reused values
    int x_center = width / 2;
    int y_center = height / 2;
    double dx = (double)(x - x_center);
    double dy = (double)(y - y_center);

    // Compute polar coordinates
    double R = 2 * sqrt(dx * dx + dy * dy);
    double theta = (atan2(dy, dx) + PI) * height / (2 * PI);

    // Calculate indices ensuring they are within bounds
    int x1 = (int)R % width;
    int y1 = (int)theta % height;
    int xp1 = (x1 + 1) % width;
    int yp1 = (y1 + 1) % height;

    // Retrieve matrix values
    double z1 = tb1[width * y1 + x1];
    double z2 = tb1[width * yp1 + x1];
    double z3 = tb1[width * yp1 + xp1];
    double z4 = tb1[width * y1 + xp1];

    // Interpolation weights
    double dx_interp = theta - floor(theta);
    double dy_interp = R - floor(R);

    // Bi-linear interpolation
    double zp = z1 + dy_interp * (z2 - z1);
    double zq = z4 + dy_interp * (z3 - z4);
    double ZR = zp + dx_interp * (zq - zp);

    // Store the result
    tb2[width * y + x] = ZR;
}