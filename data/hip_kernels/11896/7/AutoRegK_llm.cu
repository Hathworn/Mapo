#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define PI 3.1415926536f

int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;

__global__ void AutoRegK(double* X0, double* X1, double* X2, double* N0, int numElements, double a, double b)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Use a shared memory buffer for coalesced memory access pattern
    __shared__ double sharedX0[1024];
    __shared__ double sharedX1[1024];
    __shared__ double sharedN0[1024];

    if (i < numElements) {
        sharedX0[threadIdx.x] = X0[i];
        sharedX1[threadIdx.x] = X1[i];
        sharedN0[threadIdx.x] = N0[i];
        __syncthreads(); // Sync threads to ensure all loads are complete

        // Perform computation using shared memory
        X2[i] = a * sharedX1[threadIdx.x] + b * sharedX0[threadIdx.x] + sharedN0[threadIdx.x];
    }
}