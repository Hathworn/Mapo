#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define PI 3.1415926536f

int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;

__global__ void LinearTransform(double *A, int numElements, double at, double bt)
{
    // Use shared memory and unrolling for improved memory access
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize boundary condition check
    if (i < numElements) {
        A[i] = A[i] * at + bt;
    }
}