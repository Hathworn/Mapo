#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// includes, project

#define PI 3.1415926536f

int MaxThreadsPerBlock;
int MaxThreadsX;
int MaxThreadsY;

__global__ void AutoRegPinkK(double* X0, double* X1, double* Y, double* C0, double* D0, double* LastF, int numElts, int Nc)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElts) {
        double sum = 0.0; // Use a local variable to accumulate the sum for reduced memory access
        for (int j = 0; j < Nc; j++) {
            int idx = j * numElts + i;
            double temp = (X0[i] + X1[i]) * D0[j] - C0[j] * LastF[idx];
            LastF[idx] = temp; // Store the computed value back
            sum += temp; // Accumulate the total sum
        }
        Y[i] = sum; // Assign the accumulated sum to Y[i]
    }
}