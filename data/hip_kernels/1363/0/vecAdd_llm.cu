#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(int *a, int *b, int *c, int n)
{
    // Using shared memory to hold a and b sub-arrays for better performance
    extern __shared__ int sharedMemory[];
    int *sharedA = &sharedMemory[0];
    int *sharedB = &sharedMemory[blockDim.x];

    // Load data into shared memory
    int tid = threadIdx.x;
    int id = blockIdx.x * blockDim.x + tid;

    if (id < n) {
        sharedA[tid] = a[id];
        sharedB[tid] = b[id];
        __syncthreads();  // Ensure all data is loaded

        // Perform vector addition
        c[id] = sharedA[tid] + sharedB[tid];
    }
}