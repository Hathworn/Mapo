#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define MAX_STEPS 32

void readFile();
int cpuPathTest(int limitSteps, unsigned long long *tid);
void printMaze();
void printPath(unsigned long long tid, int steps);
void printPathMaze(unsigned long long tid, int steps);
void setTime0();
void getExeTime();

struct Maze {
    char maze[99][99];
    int rows, cols, s_x, s_y, e_x, e_y;
};

struct Maze maze;
FILE *MAZE;
struct timespec t_start, t_end;
double elapsedTime;

const int threadsPerBlock = 1024;
const int blocksPerGrid = 1024;

__global__ void testPath(int *limitSteps, struct Maze *maze, int *workDone, unsigned long long *path) {
    unsigned long long tid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned long long bias = blockDim.x * gridDim.x;
    unsigned long long maxRoute = 0xffffffffffffffff - (bias - 1);
    maxRoute >>= (MAX_STEPS - *limitSteps) * 2;

    while (tid <= maxRoute) {
        if (*workDone) break;

        int x = maze->s_x, y = maze->s_y;
        unsigned long long temp = tid;
        int steps = 0;
        int i = *limitSteps;

        // Use unsigned 'direction' instead of int for optimization
        unsigned direction;
        do {
            // Extract and shift only once per iteration
            direction = temp & 3; // Utilize register for better performance
            temp >>= 2;
            steps++;

            // Use a lookup table for direction increments to improve branch prediction
            const int dx[4] = {1, 0, -1, 0};
            const int dy[4] = {0, -1, 0, 1};
            x += dx[direction];
            y += dy[direction];

            // Check target condition and maze boundary in one conditional
            if (maze->maze[y][x] == '$') {
                atomicExch(workDone, 1); // Use atomic to update shared 'workDone'
                atomicExch(path, tid); // Similarly, atomic exchange for path
                break;
            } else if (maze->maze[y][x] != '.') {
                break;
            }
        } while (--i);

        tid += bias;
    }
}