#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void jackpot_compactTest_gpu_ADD(uint32_t *data, uint32_t *partial_sums, int len)
{
    __shared__ uint32_t buf;
    int id = ((blockIdx.x * blockDim.x) + threadIdx.x);

    if (id >= len) return; // Use '>=' to correctly handle boundary condition

    if (threadIdx.x == 0)
    {
        buf = partial_sums[blockIdx.x]; // Initialize shared memory with partial sums
    }

    __syncthreads();
    data[id] += buf; // Add the buffer to each data element
}