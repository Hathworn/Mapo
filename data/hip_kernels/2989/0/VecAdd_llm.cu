#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel definition

__global__ void VecAdd(float* A, float* B, float* C, int nums)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Check if index is within bounds and optimize memory access
    if (i < nums)
    {
        C[i] = A[i] + B[i];
    }
}