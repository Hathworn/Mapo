#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaProcessUnsignedChar0(unsigned char *dst, unsigned char *src, int imgW, int imgH)
{
    // Calculate global positions
    int x = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    int y = (blockIdx.y * blockDim.y + threadIdx.y) * 2;
    int px = y * imgW + x;

    // Ensure within bounds
    if (x >= imgW - 2 || y >= imgH - 2 || x < 1 || y < 1) return;

    // Precompute source indexes
    int sx1 = px - imgW;
    int sx2 = sx1 + 1;
    int sx3 = sx1 + 2;
    int sx4 = px - 1;
    int sx5 = px;
    int sx6 = px + 1;
    int sx7 = sx6 + 1;
    int sx8 = px + imgW - 1;
    int sx9 = px + imgW;
    int sxa = sx9 + 1;
    int sxb = sx9 + 2;
    int sxc = px + imgW * 2 - 1;
    int sxd = px + imgW * 2;
    int sxe = sxd + 1;

    // Load source data into local registers
    int g1 = src[sx2];
    int g2 = src[sx5];
    int g3 = src[sx7];
    int g4 = src[sx8];
    int g5 = src[sxa];
    int g6 = src[sxd];
    int b0 = src[sx4];
    int b1 = src[sx6];
    int b2 = src[sxc];
    int b3 = src[sxe];
    int r0 = src[sx1];
    int r1 = src[sx3];
    int r2 = src[sx9];
    int r3 = src[sxb];

    // Compute destination values
    int db0 = (b0 + b1) >> 1;
    int dg0 = g2;
    int dr0 = (r0 + r1) >> 1;
    int db1 = b1;
    int dg1 = (g1 + g2 + g3 + g5) >> 2;
    int dr1 = (r0 + r1 + r2 + r3) >> 2;
    int db2 = (b0 + b1 + b2 + b3) >> 2;
    int dg2 = (g2 + g4 + g5 + g6) >> 2;
    int dr2 = r2;
    int db3 = (b1 + b3) >> 1;
    int dg3 = g5;
    int dr3 = (r2 + r3) >> 1;

    // Precompute destination indexes
    int dx = px * 3;
    int dst0 = dx;
    int dst1 = dx + 3;
    int dst2 = dx + imgW * 3;
    int dst3 = dx + (imgW + 1) * 3;

    // Store computed values to destination
    dst[dst0 + 0] = (unsigned char)db0;
    dst[dst0 + 1] = (unsigned char)dg0;
    dst[dst0 + 2] = (unsigned char)dr0;
    dst[dst1 + 0] = (unsigned char)db1;
    dst[dst1 + 1] = (unsigned char)dg1;
    dst[dst1 + 2] = (unsigned char)dr1;
    dst[dst2 + 0] = (unsigned char)db2;
    dst[dst2 + 1] = (unsigned char)dg2;
    dst[dst2 + 2] = (unsigned char)dr2;
    dst[dst3 + 0] = (unsigned char)db3;
    dst[dst3 + 1] = (unsigned char)dg3;
    dst[dst3 + 2] = (unsigned char)dr3;
}