#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float *a, size_t N)
{
    int tid = threadIdx.x;
    __shared__ float s[BS];
    float sum = 0.0f;

    for (int ib = blockIdx.x; ib < (N + BS - 1) / BS; ib += gridDim.x) // Use grid-stride loop
    {
        int off = ib * BS + tid;
        s[tid] = (off < N) ? a[off] : 0.0f; // Prevent out-of-bounds access
        
        __syncthreads(); // Ensure all threads have loaded data

        for (int skip = BS / 2; skip > 0; skip >>= 1) // Perform parallel reduction
        {
            if (tid < skip)
                s[tid] += s[tid + skip];
            __syncthreads(); // Ensure all threads complete each step
        }
        
        if (tid == 0) // Only one thread updates the global sum
            sum += s[0];
    }

    if (tid == 0) // Store result from the first thread
        a[0] = sum;
}