#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimization: Use shared memory to reduce global memory access latency
extern "C"
__global__ void recipSummation(double* data, double* recip, int len)
{
    extern __shared__ double shared_recip[]; // Shared memory for recip
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalThreads = gridDim.x * blockDim.x * gridDim.y;

    for (int i = x; i < len; i += totalThreads) {
        shared_recip[threadIdx.x] = recip[i]; // Load recip into shared memory
        __syncthreads(); // Synchronize within the block

        const int j = 2 * i;
        data[j]     *= shared_recip[threadIdx.x];
        data[j + 1] *= shared_recip[threadIdx.x];
        __syncthreads(); // Ensure all threads have processed the current slice
    }
}