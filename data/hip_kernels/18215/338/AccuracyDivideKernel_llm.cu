#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread index to ensure atomic division
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        atomicExch(accuracy, *accuracy / N);
    }
}