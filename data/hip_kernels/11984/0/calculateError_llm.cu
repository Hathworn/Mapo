#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculateError(float *aFourth, float *err, int expectedOutput)
{
    // Use blockIdx and blockDim to handle more data or larger arrays
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure index is within array bounds
    if (index < expectedOutput) {
        err[index] = aFourth[index] - (index + 1 == expectedOutput);
    }
}