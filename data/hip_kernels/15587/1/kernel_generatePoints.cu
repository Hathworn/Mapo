#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_generatePoints( hiprandState * globalState, int* counts, int totalNumThreads)
{
int index = (blockIdx.x * blockDim.x) + threadIdx.x;
float x,y;
if(index >= totalNumThreads){
return;
}
hiprandState localState = globalState[index];
for(int i = 0; i < NUM_POINTS_PER_THREAD; i++)
{
x = hiprand_uniform( &localState);
y = hiprand_uniform( &localState);
if(x*x+y*y <=1){
counts[index]++;
}
}
globalState[index] = localState;
}