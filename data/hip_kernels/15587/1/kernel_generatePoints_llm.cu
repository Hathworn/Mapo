#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_generatePoints(hiprandState *globalState, int *counts, int totalNumThreads)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= totalNumThreads)
    {
        return;
    }

    // Load the state from global memory once
    hiprandState localState = globalState[index];
    int localCount = 0;
    
    for (int i = 0; i < NUM_POINTS_PER_THREAD; i++)
    {
        float x = hiprand_uniform(&localState);
        float y = hiprand_uniform(&localState);
        if (x * x + y * y <= 1)
        {
            localCount++;
        }
    }
    
    // Write the result back to global memory once
    counts[index] = localCount;
    // Store the state back to global memory once
    globalState[index] = localState;
}