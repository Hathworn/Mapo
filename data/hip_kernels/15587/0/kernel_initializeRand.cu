#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_initializeRand( hiprandState * randomGeneratorStateArray, unsigned long seed, int totalNumThreads)
{
int id = (blockIdx.x * blockDim.x) + threadIdx.x;
if( id >= totalNumThreads){
return;
}
hiprand_init( seed, id, 0, &randomGeneratorStateArray[id]);
}