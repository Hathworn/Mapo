#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_initializeRand(hiprandState *randomGeneratorStateArray, unsigned long seed, int totalNumThreads) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll the loop for better performance:
    if (id < totalNumThreads) {
        hiprand_init(seed, id, 0, &randomGeneratorStateArray[id]);
    }
}