#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c, int n) {
    // Calculate the global index.
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure no threads do unnecessary work.
    if (index < n) {
        // Perform the addition only for valid indices.
        c[index] = a[index] + b[index];
    }
}