#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initArray(uint32_t *path, double *approx, uint32_t *top_k, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use a single if statement and directly access arrays using `index`
    if (index < n) {
        approx[index]++;
        top_k[index] = path[index];
        path[index]++;
    }
}