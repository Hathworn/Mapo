#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Return early if the thread is out of bounds
    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate target pixel position and displacement
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);

    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];
    float weight;

    // Process and update all relevant pixels
    // Bottom right corner
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        weight = dx * dy;
        atomicAdd(&dst[ty * image_stride + tx], value * weight);
        atomicAdd(&normalization_factor[ty * image_stride + tx], weight);
    }

    // Bottom left corner
    tx -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        weight = (1.0f - dx) * dy;
        atomicAdd(&dst[ty * image_stride + tx], value * weight);
        atomicAdd(&normalization_factor[ty * image_stride + tx], weight);
    }

    // Upper left corner
    ty -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(&dst[ty * image_stride + tx], value * weight);
        atomicAdd(&normalization_factor[ty * image_stride + tx], weight);
    }

    // Upper right corner
    tx += 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        weight = dx * (1.0f - dy);
        atomicAdd(&dst[ty * image_stride + tx], value * weight);
        atomicAdd(&normalization_factor[ty * image_stride + tx], weight);
    }
}