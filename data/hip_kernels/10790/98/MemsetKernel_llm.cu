#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;  // Calculate global row index
    int j = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global column index

    if (i >= h || j >= w) return;  // Bounds check

    const int pos = i * w + j;  // Calculate linear memory index

    image[pos] = value;  // Set image pixel value
}