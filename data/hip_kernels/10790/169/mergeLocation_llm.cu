#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Use registers to avoid redundant memory access
        const short2 loc = loc_[ptidx];

        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}