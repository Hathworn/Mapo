#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix

extern "C"   // ensure function name to be exactly "eeTanh"
{

}

__global__ void sqErrDeriv(int N, int M, float *A, float *Y, float *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < M)  // Check bounds before calculating the index
    {
        int index = j * N + i;  // Calculate index inside bounds check
        out[index] = __fmul_rn(2.0, __fsub_rn(A[index], Y[index]));
        // Optimized calculation with fast math operations
    }
}