#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"   // ensure function name to be exactly "eeTanh"
{
}

__global__ void swap_matrix_col(int N, int C, float *X, float *V)
{
    // Use shared memory to reduce global memory accesses
    extern __shared__ float shared_data[];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int index = (C-1) * N + i;

    if (i < N)
    {
        // Load V[i] to shared memory
        shared_data[threadIdx.x] = V[i];
        __syncthreads();

        float a = X[index];

        // Use shared memory for improved access pattern
        X[index] = shared_data[threadIdx.x];
        V[i] = a;
    }
}