#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"   // ensure function name to be exactly "eeTanh"
{
}

__global__ void tanhActivation(int N, int M, float *z)
{
    // Flattened index for 2D matrix
    int index = blockIdx.y * blockDim.y * N + blockIdx.x * blockDim.x + threadIdx.y * N + threadIdx.x;

    float c1 = __fdividef(2.0, 3.0);

    if (index < N * M) // Single check for index within bounds
    {
        float el = __fmul_rn(z[index], c1);
        if (el > 4.97)
        {
            z[index] = 1.7159;
        }
        else if (el < -4.97)
        {
            z[index] = -1.7159;
        }
        else
        {
            float x2 = __fmul_rn(el, el);
            float a = __fmul_rn(el, __fmaf_rn(x2, __fmaf_rn(x2, __fadd_rn(378.0, x2), 17235.0), 135135.0));
            float b = __fmaf_rn(x2, __fmaf_rn(x2, __fmaf_rn(x2, 28.0, 3150.0), 62370.0), 135135.0);
            float tanh = __fdividef(a, b);
            z[index] = __fmul_rn(1.7159, tanh);
        }
    }
}