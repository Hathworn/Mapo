#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu

extern "C"   // ensure function name to be exactly "eeTanh"
{

}

__global__ void absErr(int N, int M, float *A, float *Y)
{
    // Efficient flat indexing for 2D grid
    int index = blockIdx.y * blockDim.y * N + blockIdx.x * blockDim.x + threadIdx.y * N + threadIdx.x;

    // Use index if within bounds
    if (index < M * N)
    {
        A[index] = fabsf(__fsub_rn(A[index], Y[index]));
    }
}