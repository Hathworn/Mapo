#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize `noactivationGradient` kernel
__global__ void noactivationGradient(int N, int M, float *z, float *tanh_grad_z, int seed, float D) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int index = j * N + i;
    if (i >= N || j >= M) return; // Boundary check at the beginning

    hiprandState_t state;
    hiprand_init((seed << 20) + index, 0, 0, &state);
    
    float F = hiprand_uniform(&state);
    float scaleFactor = __fdividef(1.0, __fsub_rn(1.0, D)); // Calculate scaleFactor once

    if (D != 0.0) {
        if (F < D) {
            z[index] = 0.0;
            tanh_grad_z[index] = 0.0;
        } else {
            tanh_grad_z[index] = scaleFactor;
            z[index] *= scaleFactor; // Use shorthand operator
        }
    } else {
        tanh_grad_z[index] = 1.0;
    }
}