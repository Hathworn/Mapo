#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix

extern "C"   // ensure function name to be exactly "eeTanh"
{
}

__global__ void finish_delta(int N, int M, float *A, float *Y, float *out)
{
    // Calculate global thread index for x and y dimensions.
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Compute flat index for row-major 2D grid.
    int index = j * N + i;

    // Ensure the thread is within bounds before performing operations.
    if (i < N && j < M)
    {
        // Compact and efficient sign computation.
        out[index] = copysignf(1.0, A[index] - Y[index]);
    }
}