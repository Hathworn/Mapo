#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
{

}

__global__ void elSq2(int N, int M, float *In, float *Out)
{
    // Use fast math operation
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < M) {
        // Directly calculate index only when needed
        int index = j * N + i;
        Out[index] = __fmul_rn(In[index], In[index]);
    }
}