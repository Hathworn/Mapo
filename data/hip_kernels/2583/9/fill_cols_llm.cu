#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"   // ensure function name to be exactly "eeTanh"
{

}

__global__ void fill_cols(int N, int M, float *X, float *V)
{
    // Calculate the index only once
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < M)
    {
        X[j * N + i] = V[j]; // Use pre-computed index
    }
}