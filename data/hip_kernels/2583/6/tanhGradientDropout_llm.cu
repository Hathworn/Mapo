#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"   // ensure function name to be exactly "eeTanh"
{
}

__global__ void tanhGradientDropout(int N, int M, float *z, float *tanh_grad_z, int seed, float D) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= N || j >= M) return;  // Efficient boundary check

    int index = j * N + i;
    float c1 = __fdividef(2.0, 3.0);
    float scaleFactor1 = __fdividef(1.7159, __fsub_rn(1.0, D));
    float scaleFactor2 = __fdividef(-1.7159, __fsub_rn(1.0, D));

    // Initialize random state
    hiprandState_t state;
    hiprand_init((seed << 20) + index, 0, 0, &state);

    // Generate a uniform random number
    float F = hiprand_uniform(&state);

    if (F < D) {
        z[index] = 0.0;
        tanh_grad_z[index] = 0.0;
    } else {
        float el = __fmul_rn(z[index], c1);
        if (el > 4.97) {
            z[index] = scaleFactor1;
            tanh_grad_z[index] = 0.0;
        } else if (el < -4.97) {
            z[index] = scaleFactor2;
            tanh_grad_z[index] = 0.0;
        } else {
            float x2 = __fmul_rn(el, el);
            float a = __fmul_rn(el, __fmaf_rn(x2, __fmaf_rn(x2, __fadd_rn(378.0, x2), 17235.0), 135135.0));
            float b = __fmaf_rn(x2, __fmaf_rn(x2, __fmaf_rn(x2, 28.0, 3150.0), 62370.0), 135135.0);
            float tanh = __fdividef(a, b);
            z[index] = __fmul_rn(scaleFactor1, tanh);
            tanh_grad_z[index] = __fmul_rn(scaleFactor1, __fmul_rn(__fmaf_rn(-tanh, tanh, 1.0), c1));
        }
    }
}