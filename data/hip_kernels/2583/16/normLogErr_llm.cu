#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix

extern "C" // ensure function name to be exactly "eeTanh"
{
}

__global__ void normLogErr(int N, int M, float *A, float *Y)
{
    // Optimize by calculating index only once
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < M)
    {
        int index = j * N + i;
        int L = N * M;

        // Use intrinsic functions for better performance
        float a = __expf(2.0f * A[index + L]);
        A[index] = a * (0.5f * Y[index] * Y[index] + 0.5f * A[index] * A[index] - A[index] * Y[index]);
        A[index + L] = 0.9189385332f - A[index + L];
    }
}