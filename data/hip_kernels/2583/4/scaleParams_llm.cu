#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
{

}

__global__ void scaleParams(int N, int M, float c, float *Mat, float *F)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Check indices once to improve readability and reduce code duplication
    if (i < N && j < M)
    {
        int index = j * N + i; // Move index calculation inside the if condition
        float s = __saturatef(__fdividef(c, __fsqrt_rn(F[i])));
        Mat[index] = __fmul_rn(Mat[index], s);
    }
}