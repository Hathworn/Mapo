#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elMul(int N, int M, float *X1, float *X2)
{
    // Efficiently compute global thread index using built-in variables
    int index = blockIdx.y * blockDim.y * N + blockIdx.x * blockDim.x + threadIdx.y * N + threadIdx.x;

    // Check if index is within bounds and perform multiplication
    if (index < N * M)
    {
        X1[index] = __fmul_rn(X1[index], X2[index]); // Use fast multiply operation
    }
}