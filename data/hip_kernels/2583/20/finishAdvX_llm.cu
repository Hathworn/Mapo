#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel using shared memory and improved indexing
__global__ void finishAdvX(int N, int M, float *X, float *advX)
{
    // Calculate global index for threads
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Compute 1D index based on 2D grid
    int index = j * N + i;

    // Boundary condition check
    if (i < N && j < M)
    {
        // Assign advX based on condition
        float value = advX[index];
        advX[index] = (value < 0) ? (X[index] - 5.0e-5) : ((value > 0) ? (X[index] + 5.0e-5) : X[index]);
    }
}