#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"   // ensure function name to be exactly "eeTanh"
{
}

__global__ void tanhGradient(int N, int M, float *z, float *tanh_grad_z) {
    // Calculate global indices for the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < M) { // Boundary check
        int index = j * N + i;

        // Constants
        float c1 = __fdividef(2.0, 3.0);
        float c2 = 1.7159;

        // Compute and optimize computations within branches
        float el = __fmul_rn(z[index], c1);

        if (el > 4.97) {
            z[index] = c2;
            tanh_grad_z[index] = 0.0f;
        }
        else if(el < -4.97) {
            z[index] = -c2;
            tanh_grad_z[index] = 0.0f;
        }
        else {
            float x2 = __fmul_rn(el, el);
            float a = __fmul_rn(el, __fmaf_rn(x2, __fmaf_rn(x2, __fadd_rn(378.0, x2), 17235.0), 135135.0));
            float b = __fmaf_rn(x2, __fmaf_rn(x2, __fmaf_rn(x2, 28.0, 3150.0), 62370.0), 135135.0);
            float tanh_val = __fdividef(a, b);
            z[index] = __fmul_rn(c2, tanh_val);
            tanh_grad_z[index] = __fmul_rn(c2, __fmul_rn(__fmaf_rn(-tanh_val, tanh_val, 1.0f), c1));
        }
    }
}