#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
{

}

__global__ void elSq(int N, int M, float *Mat)
{
    // Calculate global thread index for single dimension access
    int idx = blockIdx.y * blockDim.y * N + blockIdx.x * blockDim.x + threadIdx.y * N + threadIdx.x;

    // Guard against out-of-bounds and perform element-wise squaring
    if (threadIdx.x + blockIdx.x * blockDim.x < N && threadIdx.y + blockIdx.y * blockDim.y < M)
    {
        Mat[idx] = __fmul_rn(Mat[idx], Mat[idx]);
    }
}