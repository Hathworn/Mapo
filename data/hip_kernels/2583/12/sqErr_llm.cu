#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// filename: eeTanh.cu

extern "C" // ensure function name to be exactly "eeTanh"
{

}

__global__ void sqErr(int N, int M, float *A, float *Y)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Check boundaries to avoid out-of-bounds access
    if (i < N && j < M)
    {
        int index = j * N + i;  // Compute 1D index
        float tmp = __fsub_rn(A[index], Y[index]);  // Use fast subtraction
        A[index] = __fmul_rn(tmp, tmp);  // Use fast multiplication for squaring
    }
}