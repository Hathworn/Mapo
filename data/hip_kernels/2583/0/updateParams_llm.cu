#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
{










}
__global__ void updateParams(int N, int M, float alpha, float beta1, float beta2, float t, float *PARAMS, float *GRADS, float *m, float *v)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate linear index once for reuse
    int index = j * N + i;

    if (i < N && j < M)
    {
        // Precompute to avoid redundant calculation
        float beta1r = 1.0f - beta1;
        float alphar = -alpha / (1.0f - __powf(beta1, t));

        // Update moving average of gradients
        float grad = GRADS[index];
        float m_val = m[index] = beta1 * m[index] + beta1r * grad;

        // Update moving average of squared gradients with safe guard
        v[index] = max(beta2 * v[index], max(fabsf(grad), 1.0e-16f));

        // Update parameters leveraging precomputed alphar
        PARAMS[index] = alphar * (m_val / v[index]) + PARAMS[index];
    }
}