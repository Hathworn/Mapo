#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
{

}

__global__ void updateEst(int N, int M, float beta2, float scale, float *PARAMS, float *AVG, float *EST)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < N && j < M)
    {
        int index = j * N + i;
        // Use fused multiply-add for better performance
        AVG[index] = __fmaf_rn((1.0f - beta2), PARAMS[index], __fmul_rn(beta2, AVG[index]));
        EST[index] = __fmul_rn(scale, AVG[index]);
    }
}