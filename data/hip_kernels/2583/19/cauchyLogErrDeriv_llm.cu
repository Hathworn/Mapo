#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
{
}

__global__ void cauchyLogErrDeriv(int N, int M, float *A, float *Y, float *out)
{
    // Use shared memory for faster access (assuming N and M are small)
    __shared__ float sharedA[1024]; // Adjust size based on shared memory usage constraints
    __shared__ float sharedY[1024]; // Adjust size based on shared memory usage constraints

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int index = j*N + i;
    int L = N*M;

    if (i < N && j < M)
    {
        // Load data into shared memory
        sharedA[threadIdx.y * blockDim.x + threadIdx.x] = A[index];
        sharedY[threadIdx.y * blockDim.x + threadIdx.x] = Y[index];
        __syncthreads();

        // Use shared memory for computation
        float a = __expf(sharedA[threadIdx.y * blockDim.x + threadIdx.x + L]);
        
        if (sharedA[threadIdx.y * blockDim.x + threadIdx.x] > sharedY[threadIdx.y * blockDim.x + threadIdx.x])
        {
            out[index] = a;
        }
        else if (sharedA[threadIdx.y * blockDim.x + threadIdx.x] < sharedY[threadIdx.y * blockDim.x + threadIdx.x])
        {
            out[index] = -a;
        }
        else
        {
            out[index] = 0.0;
        }

        out[index+L] = __fmaf_rn(a, fabsf(__fsub_rn(sharedA[threadIdx.y * blockDim.x + threadIdx.x], sharedY[threadIdx.y * blockDim.x + threadIdx.x])), -1.0);
    }
}