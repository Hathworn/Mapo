#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// filename: eeTanh.cu
// a simple CUDA kernel to square the elements of a matrix

extern "C" // ensure function name to be exactly "eeTanh"
{

__global__ void absErrDeriv(int N, int M, float *A, float *Y, float *out)
{
    // Use shared memory for better memory access efficiency
    extern __shared__ float sharedMem[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate global index
    int index = j * N + i;

    // Check boundaries
    if (i < N && j < M)
    {
        // Use registers and shared memory for inputs within block
        float a_val = A[index];
        float y_val = Y[index];
        sharedMem[threadIdx.y * blockDim.x + threadIdx.x] = copysignf(1.0, __fsub_rn(a_val, y_val));

        __syncthreads();

        // Write output back from shared memory
        out[index] = sharedMem[threadIdx.y * blockDim.x + threadIdx.x];
    }
}

}