#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FillOnes( float *vec, int size ) {
    // Use a stride loop to handle larger arrays efficiently
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < size; idx += gridDim.x * blockDim.x) {
        vec[idx] = 1.0f;
    }
}