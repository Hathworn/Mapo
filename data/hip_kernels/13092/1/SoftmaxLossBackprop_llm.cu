#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SoftmaxLossBackprop(const float *label, int num_labels, int batch_size, float *diff) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Efficient boundary check
    if (idx < batch_size) {
        const int label_value = static_cast<int>(label[idx]);
        // Perform atomic operation to avoid race condition
        atomicSub(&diff[idx * num_labels + label_value], 1.0f);
    }
}