#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelCalculateHistogram(unsigned int* histogram, unsigned char* rawPixels, long chunkSize, long totalPixels)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    // Calculate stride based on total number of threads
    int stride = blockDim.x * gridDim.x; 

    // Loop with stride to allow all threads to participate
    for (int i = id * chunkSize; i < totalPixels; i += stride * chunkSize) {
        int end = min(i + chunkSize, totalPixels);
        for (int j = i; j < end; j++) {
            int pixelValue = (int)rawPixels[j];
            atomicAdd(&histogram[pixelValue], 1);
        }
    }
}