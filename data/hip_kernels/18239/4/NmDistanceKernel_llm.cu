#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NmDistanceKernel(int b, int n, const float *xyz, int m, const float *xyz2, float *result, int *result_i) {
    const int batch = 512;
    __shared__ float buf[batch * 3];

    // Loop over batches
    for (int i = blockIdx.x; i < b; i += gridDim.x) {
        for (int k2 = 0; k2 < m; k2 += batch) {
            // Load current batch into shared memory
            int end_k = min(m, k2 + batch) - k2;
            for (int j = threadIdx.x; j < end_k * 3; j += blockDim.x) {
                buf[j] = xyz2[(i * m + k2) * 3 + j];
            }
            __syncthreads();

            // Compute distances for each point
            for (int j = threadIdx.x + blockIdx.y * blockDim.x; j < n; j += blockDim.x * gridDim.y) {
                float x1 = xyz[(i * n + j) * 3 + 0];
                float y1 = xyz[(i * n + j) * 3 + 1];
                float z1 = xyz[(i * n + j) * 3 + 2];
                int best_i = 0;
                float best = INFINITY;  // Use INFINITY for initialization for better numerical safety

                // Loop unrolling for batch calculations
                int end_ka = end_k - (end_k & 3);
                for (int k = 0; k < end_ka; k += 4) {
                    #pragma unroll
                    for (int offset = 0; offset < 4; ++offset) {
                        float x2 = buf[(k + offset) * 3 + 0] - x1;
                        float y2 = buf[(k + offset) * 3 + 1] - y1;
                        float z2 = buf[(k + offset) * 3 + 2] - z1;
                        float d = x2 * x2 + y2 * y2 + z2 * z2;
                        if (k == 0 || d < best) {
                            best = d;
                            best_i = k + offset + k2;
                        }
                    }
                }

                // Handle the remaining cases if any
                for (int k = end_ka; k < end_k; k++) {
                    float x2 = buf[k * 3 + 0] - x1;
                    float y2 = buf[k * 3 + 1] - y1;
                    float z2 = buf[k * 3 + 2] - z1;
                    float d = x2 * x2 + y2 * y2 + z2 * z2;
                    if (d < best) {
                        best = d;
                        best_i = k + k2;
                    }
                }

                // Update result only if the current best is better
                if (k2 == 0 || result[(i * n + j)] > best) {
                    result[(i * n + j)] = best;
                    result_i[(i * n + j)] = best_i;
                }
            }
            __syncthreads();
        }
    }
}