#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define inf 9999

__global__ void funct(int n, int k, float* x, int* qx)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= n * n) return; // Early exit for out-of-bounds threads

    int j = ix & (n - 1); // Optimized index computation
    float temp2 = x[ix - j + k] + x[k * n + j];

    if (x[ix] > temp2) {
        x[ix] = temp2;
        qx[ix] = k;
    }

    if (x[ix] == inf) {
        qx[ix] = -2;
    }
}