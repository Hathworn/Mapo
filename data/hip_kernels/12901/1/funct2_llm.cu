#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/* Kintsakis Athanasios AEM 6667 */

#define inf 9999

__global__ void funct2(int n, int k, float* x, int* qx)
{
    // Calculate global thread index
    int ix = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure memory access is within bounds
    if (ix < n * n) {
        int j = ix & (n - 1); // Calculate column index
        float temp2 = x[ix - j + k] + x[k * n + j]; // Common subexpression
        if (x[ix] > temp2) { // Compare and update
            x[ix] = temp2;
            qx[ix] = k;
        }
    }
}