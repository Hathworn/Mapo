#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// GPU constant memory to hold our kernels (extremely fast access time)
__constant__ float convolutionKernelStore[256];

// Kernel function optimized for shared memory usage and memory access patterns
__global__ void convolve(unsigned char *source, int width, int height, int paddingX, int paddingY, size_t kOffset, int kWidth, int kHeight, unsigned char *destination)
{
    // Calculate pixel position
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Use shared memory for storing kernel values
    __shared__ float shKernel[256]; // Adjust the size according to maximum kernel size
    if (threadIdx.x < kWidth && threadIdx.y < kHeight) {
        int kIndex = threadIdx.y * kWidth + threadIdx.x;
        shKernel[kIndex] = convolutionKernelStore[kIndex + kOffset];
    }
    __syncthreads();

    float sum = 0.0f;
    int pWidth = kWidth / 2;
    int pHeight = kHeight / 2;

    // Process only valid pixels
    if (x >= pWidth + paddingX &&
        y >= pHeight + paddingY &&
        x < (blockDim.x * gridDim.x) - pWidth - paddingX &&
        y < (blockDim.y * gridDim.y) - pHeight - paddingY)
    {
        for (int j = -pHeight; j <= pHeight; ++j)
        {
            for (int i = -pWidth; i <= pWidth; ++i)
            {
                int ki = (i + pWidth);
                int kj = (j + pHeight);
                // Access shared memory for kernel weights
                float w = shKernel[(kj * kWidth) + ki];
                sum += w * float(source[((y + j) * width) + (x + i)]);
            }
        }
    }
    // Average the sum and store in destination
    destination[(y * width) + x] = (unsigned char)sum;
}