#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// GPU constant memory to hold our kernels (extremely fast access time)
__constant__ float convolutionKernelStore[256];

/**
* Convolution function for cuda.  Destination is expected to have the same width/height as source, but there will be a border
* of floor(kWidth/2) pixels left and right and floor(kHeight/2) pixels top and bottom
*
* @param source      Source image host pinned memory pointer
* @param width       Source image width
* @param height      Source image height
* @param paddingX    source image padding along x
* @param paddingY    source image padding along y
* @param kOffset     offset into kernel store constant memory
* @param kWidth      kernel width
* @param kHeight     kernel height
* @param destination Destination image host pinned memory pointer
*/

// Optimization: Use shared memory for better performance
__global__ void pythagoras(unsigned char *a, unsigned char *b, unsigned char *c)
{
    extern __shared__ float s_mem[];
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Load data into shared memory
    s_mem[threadIdx.x] = float(a[idx]);
    __syncthreads();

    float af = s_mem[threadIdx.x];
    float bf = float(b[idx]);

    c[idx] = (unsigned char)sqrtf(af * af + bf * bf);
}