#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index is within bounds
    if (ptidx >= npoints) return;

    // Use local variables to reduce memory access latency
    short2 loc = loc_[ptidx];
    float locx = loc.x * scale;
    float locy = loc.y * scale;
    
    x[ptidx] = locx;
    y[ptidx] = locy;
}