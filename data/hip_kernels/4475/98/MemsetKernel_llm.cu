#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate 1D index from 2D grid
    int pos = (blockIdx.y * blockDim.y + threadIdx.y) * w + (blockIdx.x * blockDim.x + threadIdx.x);

    // Flattened boundary check
    if (pos < w * h) 
    {
        image[pos] = value;
    }
}