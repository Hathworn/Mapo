#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Compute index based on current thread and block
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Early exit if thread is out of bounds
    if (i >= h || j >= w) return;

    // Compute offsets only once
    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Read u and v values from input arrays
    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Calculate position of target pixel's bottom left corner
    float cx = u_ * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v_ * time_scale + static_cast<float>(i) + 1.0f;

    // Calculate pixel indices using round-to-nearest
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Read source image value
    float value = src[image_row_offset + j];

    // Safely add value to destination if pixel is within bounds
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}