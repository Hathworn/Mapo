#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate target pixel coordinates
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];
    
    // Pre-calculate weights
    float w_bottom_right = dx * dy;
    float w_bottom_left = (1.0f - dx) * dy;
    float w_upper_left = (1.0f - dx) * (1.0f - dy);
    float w_upper_right = dx * (1.0f - dy);

    // Fill pixel containing bottom right corner
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0))
    {
        _atomicAdd(dst + ty * image_stride + tx, value * w_bottom_right);
        _atomicAdd(normalization_factor + ty * image_stride + tx, w_bottom_right);
    }

    // Fill pixel containing bottom left corner
    tx -= 1;
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0))
    {
        _atomicAdd(dst + ty * image_stride + tx, value * w_bottom_left);
        _atomicAdd(normalization_factor + ty * image_stride + tx, w_bottom_left);
    }

    // Fill pixel containing upper left corner
    ty -= 1;
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0))
    {
        _atomicAdd(dst + ty * image_stride + tx, value * w_upper_left);
        _atomicAdd(normalization_factor + ty * image_stride + tx, w_upper_left);
    }

    // Fill pixel containing upper right corner
    tx += 1;
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0))
    {
        _atomicAdd(dst + ty * image_stride + tx, value * w_upper_right);
        _atomicAdd(normalization_factor + ty * image_stride + tx, w_upper_right);
    }
}