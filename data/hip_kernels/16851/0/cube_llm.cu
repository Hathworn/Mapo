#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void cube(long *deviceOutput, long *deviceInput)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    long f = deviceInput[idx];
    deviceOutput[idx] = f * f * f; // Compute cube
}