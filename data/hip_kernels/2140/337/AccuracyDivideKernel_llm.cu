#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load the accuracy value into a register for faster access
    float local_accuracy = *accuracy;

    // Perform division
    local_accuracy /= N;

    // Write the result back to global memory
    *accuracy = local_accuracy;
}