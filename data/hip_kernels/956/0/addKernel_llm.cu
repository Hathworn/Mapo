#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
# pragma warning (disable:4819)

#define ARRAYSIZE 5

__global__ void addKernel(int *c, const int *a, const int *b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < ARRAYSIZE) { // Ensure index is within bounds
        c[i] = a[i] + b[i];
    }
}