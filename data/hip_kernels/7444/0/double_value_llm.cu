#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void double_value(double *x, double *y)
{
    // Use blockIdx and blockDim to handle larger arrays
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Assuming grid is launched with enough threads
    y[idx] = 2.0 * x[idx];
}