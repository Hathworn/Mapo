#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeSquare(int *d_in, int *d_out) {
    // Optimize to use shared memory
    __shared__ int shared_in[256]; // Assuming the max block size is 256
    int index = threadIdx.x;
    shared_in[index] = d_in[index];
    __syncthreads();
    d_out[index] = shared_in[index] * shared_in[index];
}