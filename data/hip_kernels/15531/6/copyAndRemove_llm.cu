#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* G2S
* Copyright (C) 2018, Mathieu Gravey (gravey.mathieu@gmail.com) and UNIL (University of Lausanne)
*
* This program is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

//To use only for debugging purpose


#define PARTIAL_FFT

#ifndef FFTW_PLAN_OPTION
//FFTW_PATIENT
#define FFTW_PLAN_OPTION FFTW_ESTIMATE
#endif
// #if __cilk
//	#define fillVectorized(name, begin, amount, value) name[begin:amount]=value;
// #else
#define fillVectorized(name, begin, amount, value) std::fill(name+begin,name+begin+amount,value);
// #endif

__global__ void copyAndRemove(float* errosArray, unsigned int* _encodedPosition_d, float* _mismatch_d, const unsigned int i, const float val) {
    unsigned int pos = _encodedPosition_d[i] - 1; // Load encoded position once and decrement
    _encodedPosition_d[i] = pos;
    if (_mismatch_d) {
        _mismatch_d[i] = errosArray[pos]; // Use the updated position
    }
    errosArray[pos] = val; // Set value in errosArray
}