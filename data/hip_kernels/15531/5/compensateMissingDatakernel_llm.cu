#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* G2S
* Copyright (C) 2018, Mathieu Gravey (gravey.mathieu@gmail.com) and UNIL (University of Lausanne)
*
* This program is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

//To use only for debugging purpose

#define PARTIAL_FFT

#ifndef FFTW_PLAN_OPTION
//FFTW_PATIENT
#define FFTW_PLAN_OPTION FFTW_ESTIMATE
#endif
// #if __cilk
// 	#define fillVectorized(name, begin, amount, value) name[begin:amount]=value;
// #else
#define fillVectorized(name, begin, amount, value) std::fill(name+begin,name+begin+amount,value);
// #endif

__global__ void compensateMissingDatakernel(float* errosArray, float* crossErrosArray, const unsigned int size, float val) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory for crossErrosArray
    extern __shared__ float sharedCrossErrors[];
    if (j < size) {
        sharedCrossErrors[j] = crossErrosArray[j];
        __syncthreads(); // Synchronize threads to make sure shared memory is populated

        float crossError = sharedCrossErrors[j];
        if(crossError != 0.0f) {
            errosArray[j] = -std::fabs(errosArray[j] / (crossError * crossError * crossError * crossError));
        } else {
            errosArray[j] = val;
        }
    }
}