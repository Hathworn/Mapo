#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* G2S
* Copyright (C) 2018, Mathieu Gravey (gravey.mathieu@gmail.com) and UNIL (University of Lausanne)
*
* This program is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#define PARTIAL_FFT

#ifndef FFTW_PLAN_OPTION
//FFTW_PATIENT
#define FFTW_PLAN_OPTION FFTW_ESTIMATE
#endif

#define fillVectorized(name, begin, amount, value) std::fill(name+begin,name+begin+amount,value);

__global__ void fma(float* realSpace, const unsigned int size, const float alpha, const float delta) {
    // Use shared memory for improved performance when accessing frequently used values
    __shared__ float sharedAlpha;
    __shared__ float sharedDelta;
    if (threadIdx.x == 0) {
        sharedAlpha = alpha; 
        sharedDelta = delta;
    }
    __syncthreads();

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        // Avoid redundant memory access within the kernel execution
        float temp = realSpace[i];
        realSpace[i] = fmaf(temp, sharedAlpha, sharedDelta);
    }
}