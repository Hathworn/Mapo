#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* G2S
* Copyright (C) 2018, Mathieu Gravey (gravey.mathieu@gmail.com) and UNIL (University of Lausanne)
*
* This program is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

//To use only for debugging purpose

#define PARTIAL_FFT

#ifndef FFTW_PLAN_OPTION
#define FFTW_PLAN_OPTION FFTW_ESTIMATE
#endif

#define fillVectorized(name, begin, amount, value) std::fill(name+begin,name+begin+amount,value);

__global__ void setConditionement(unsigned size, unsigned* listIndex, float* listValueAtIndex, float* realSpaceArray, unsigned nbVar, unsigned var) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < size) {
        // Load elements from listIndex and listValueAtIndex into registers
        unsigned index = listIndex[j];
        float value = listValueAtIndex[j * nbVar + var];
        // Store value in realSpaceArray
        realSpaceArray[index] = value;
    }
}