#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* G2S
* Copyright (C) 2018, Mathieu Gravey (gravey.mathieu@gmail.com) and UNIL (University of Lausanne)
*
* This program is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License.
*
* This program is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#define PARTIAL_FFT

#ifndef FFTW_PLAN_OPTION
#define FFTW_PLAN_OPTION FFTW_ESTIMATE
#endif

#define fillVectorized(name, begin, amount, value) std::fill(name+begin,name+begin+amount,value);

__global__ void updateMask(float* realSpace1, float* realSpace2, const unsigned int size, const unsigned int deltaCross){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory to minimize global memory access
    __shared__ float sharedRealSpace2[256]; // Adjust size as necessary

    if (threadIdx.x < blockDim.x) {
        sharedRealSpace2[threadIdx.x] = realSpace2[(threadIdx.x + deltaCross) % size];
    }
    __syncthreads();

    if (i < size)
        realSpace1[i] *= sharedRealSpace2[i % blockDim.x];
}