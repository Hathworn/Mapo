#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_2d_mul_float_gpu(float *A, float *B, float *C, int num_rows_A, int num_cols_A, int num_cols_B) {
    // Create shared memory (per block)
    __shared__ float A_tile[N_THREADS][N_THREADS];
    __shared__ float B_tile[N_THREADS][N_THREADS];
    
    // Block index
    int bx = blockIdx.x; 
    int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x; 
    int ty = threadIdx.y;

    // Starting points for sub-matrix multiplication
    int aBegin = num_cols_A * N_THREADS * by;
    int aEnd   = aBegin + num_cols_A;
    int aStep  = N_THREADS;
    
    int bBegin = N_THREADS * bx;
    int bStep  = N_THREADS * num_cols_B;

    float sum = 0.0f;
    
    // Loop over all sub-matrices
    for (int a = aBegin, b = bBegin; a < aEnd; a += aStep, b += bStep) {
        // Collaborative loading of A and B sub-matrices into shared memory
        A_tile[ty][tx] = A[a + num_cols_A * ty + tx];
        B_tile[ty][tx] = B[b + num_cols_B * tx + ty];
        
        // Synchronize to ensure data is loaded
        __syncthreads();

        // Calculate partial results using the loaded tiles
        for (int k = 0; k < N_THREADS; ++k) {
            sum += A_tile[ty][k] * B_tile[k][tx];
        }
        
        // Synchronize before loading new sub-matrices
        __syncthreads();
    }

    // Write the result to the output matrix C
    int c = num_cols_B * N_THREADS * by + N_THREADS * bx;
    C[c + num_cols_B * ty + tx] = sum;
}