#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addSingleThread(int n, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    int stride = gridDim.x * blockDim.x; // Calculate stride

    for (int i = index; i < n; i += stride) // Use stride to process larger arrays
    {
        y[i] = x[i] + y[i];
    }
}