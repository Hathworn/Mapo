#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addMoreThreads(int n, float *x, float *y)
{
    // Use global thread index for better parallelism
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop over the array with increased stride
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}