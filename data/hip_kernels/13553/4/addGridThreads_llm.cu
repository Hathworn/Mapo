#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addGridThreads(int n, float *x, float *y)
{
    // Cache frequently used values to registers
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = index; i < n; i += stride)
    {
        // Optimize memory access pattern by coalescing
        y[i] = x[i] + y[i];
    }
}