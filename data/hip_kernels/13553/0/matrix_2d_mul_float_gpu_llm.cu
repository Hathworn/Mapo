#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N_THREADS 32

int num_rows_A = 2000; int num_rows_B = 2000; int num_rows_C = 2000;
int num_cols_A = 2000; int num_cols_B = 600; int num_cols_C = 600;

float *A = (float*) malloc(sizeof(float) * num_rows_A * num_cols_A);
float *B = (float*) malloc(sizeof(float) * num_rows_B * num_cols_B);
float *C = (float*) malloc(sizeof(float) * num_rows_C * num_cols_C);
float *C_ref = (float*) malloc(sizeof(float) * num_rows_C * num_cols_C);

__global__ void matrix_2d_mul_float_gpu(float *A, float *B, float *C, int num_rows_A, int num_cols_A, int num_cols_B) {
    // Calculate row and column index for the element
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure indices are within bounds
    if (i >= num_rows_A || k >= num_cols_B) return;

    float sum = 0.0f;

    // Perform multiplication and accumulation
    for (int j = 0; j < num_cols_A; j++) {
        sum += A[i * num_cols_A + j] * B[j * num_cols_B + k];
    }

    // Store the result in matrix C
    C[i * num_cols_B + k] = sum;
}