#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square_array(float *a, int N)
{
    // Using a more efficient loop by calculating the total number of threads
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = idx; i < N; i += stride) {
        a[i] *= a[i];
    }
}