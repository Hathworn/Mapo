#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findMax(int *m, int *cs, int n)
{
    // Calculate the global index
    int colnum = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Check if the index is within bounds
    if (colnum < n)
    {
        int max = m[colnum];

        // Traverse through each row and find the maximum
        for (int k = 1; k < n; k++)
        {
            int value = m[colnum + n * k];
            if (value > max)
            {
                max = value;
            }
        }

        // Store the maximum value in the result array
        cs[colnum] = max;
    }
}