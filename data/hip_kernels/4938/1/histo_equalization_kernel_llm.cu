#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_equalization_kernel(unsigned char *buffer, long size, int *histo, unsigned char *output) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;

    // Use shared memory to reduce global memory access
    __shared__ unsigned char sharedLUT[256];
    if (threadIdx.x < 256) {
        sharedLUT[threadIdx.x] = dev_lut[threadIdx.x];
    }
    __syncthreads();

    while (i < size) {
        // Use shared LUT data for output, reducing global memory reads
        output[i] = sharedLUT[buffer[i]] > 255 ? 255 : sharedLUT[buffer[i]];
        i += offset;
    }
}