#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernel (unsigned char *buffer, long size, int *histo)
{
    __shared__ int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;

    // Loop unrolling for better performance
    while (i + offset * 3 < size)
    {
        atomicAdd(&temp[buffer[i]], 1);
        atomicAdd(&temp[buffer[i + offset]], 1);
        atomicAdd(&temp[buffer[i + offset * 2]], 1);
        atomicAdd(&temp[buffer[i + offset * 3]], 1);
        i += offset * 4;
    }

    while (i < size)
    {
        atomicAdd(&temp[buffer[i]], 1);
        i += offset;
    }
    __syncthreads();

    // Ensure coalesced memory access
    atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
}