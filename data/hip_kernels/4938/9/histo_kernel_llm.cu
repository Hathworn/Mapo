#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernel ( unsigned char *buffer, long size, int *histo )
{
    // Use shared memory for temporary histogram accumulation
    __shared__ int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;
    // Unroll loop to improve memory access efficiency
    while (i < size)
    {
        atomicAdd( &temp[buffer[i]], 1);
        i += offset;
    }
    __syncthreads();

    // Single atomic add to global histogram
    atomicAdd( &(histo[threadIdx.x]), temp[threadIdx.x] );
}