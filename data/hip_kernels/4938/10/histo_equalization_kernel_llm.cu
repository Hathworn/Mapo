#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_equalization_kernel(unsigned char *buffer, long size, int *histo, unsigned char *output) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;

    // Use shared memory for dev_lut for faster access
    __shared__ int shared_dev_lut[256]; 
    if (threadIdx.x < 256) {
        shared_dev_lut[threadIdx.x] = dev_lut[threadIdx.x];
    }
    __syncthreads();

    // Loop unrolling to improve performance
    while (i < size) {
        #pragma unroll 4
        for (int j = 0; j < 4 && i < size; ++j, i += offset) {
            unsigned char value = buffer[i];
            output[i] = (unsigned char)(shared_dev_lut[value] > 255 ? 255 : shared_dev_lut[value]);
        }
    }
}