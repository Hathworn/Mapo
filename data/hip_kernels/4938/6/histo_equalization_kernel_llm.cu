#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histo_equalization_kernel ( unsigned char *buffer, long size, int *histo, unsigned char *output ) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;

    // Loop unrolling for improved performance
    while (i < size) {
        unsigned char pixel_value = buffer[i];
        unsigned char result = (unsigned char)(dev_lut[pixel_value] > 255 ? 255 : dev_lut[pixel_value]);
        output[i] = result;

        if (i + offset < size) {
            pixel_value = buffer[i + offset];
            result = (unsigned char)(dev_lut[pixel_value] > 255 ? 255 : dev_lut[pixel_value]);
            output[i + offset] = result;
        }

        i += 2 * offset;
    }
}