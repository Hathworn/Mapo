#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernel_2 (unsigned char *buffer, int img_w, int img_h, int *histo)
{
    int id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int id_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds to ensure no out-of-bounds access
    if (id_x < img_w && id_y < img_h) {
        atomicAdd(&histo[buffer[id_y * img_w + id_x]], 1);
    }
}