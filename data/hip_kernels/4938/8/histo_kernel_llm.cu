#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernel (unsigned char *buffer, long size, int *histo) {
    __shared__ int temp[256];
    
    // Initialize shared memory only for used elements
    if (threadIdx.x < 256)
        temp[threadIdx.x] = 0;

    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;

    // Loop unrolling to increase instruction throughput
    while (i < size) {
        atomicAdd(&temp[buffer[i]], 1);
        if (i + offset < size) atomicAdd(&temp[buffer[i + offset]], 1);
        i += 2 * offset;
    }
    
    __syncthreads();

    // Ensure atomic operation updates histo properly
    if (threadIdx.x < 256)
        atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
}