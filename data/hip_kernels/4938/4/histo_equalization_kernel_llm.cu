#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_equalization_kernel(unsigned char *buffer, long size, int *histo, unsigned char *output) {
    // Calculate global index for the thread
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // Iterate over indices processed by this thread
    for (; i < size; i += blockDim.x * gridDim.x) {
        unsigned char lut_value = (unsigned char) dev_lut[buffer[i]];
        // Use min function for boundary check and assignment
        output[i] = min(lut_value, (unsigned char)255);
    }
}