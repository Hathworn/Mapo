#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernel(unsigned char *buffer, long size, int *histo)
{
    __shared__ int temp[256];
    // Initialize shared memory for each thread block
    temp[threadIdx.x] = 0;
    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;
    
    // Iterate over the input buffer with a stride equal to the total number of threads
    while (i < size)
    {
        atomicAdd(&temp[buffer[i]], 1); // Use shared memory for local histograms
        i += offset;
    }
    __syncthreads();

    // Accumulate local histograms into the global histogram
    atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
}