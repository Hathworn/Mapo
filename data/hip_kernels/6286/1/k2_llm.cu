#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void k2(int *Aux, int *S) {
    int idx = threadIdx.x; // Compute the index only once
    Aux[idx] = S[(idx + 1) * B - 1];
}