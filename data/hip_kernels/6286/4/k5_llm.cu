#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void k5(int *Aux, int *S) {
    int tid = threadIdx.x;
    if(tid == 0) return;
    // Use shared memory for coalesced access
    __shared__ int sharedAux[B];
    sharedAux[tid] = Aux[tid];
    __syncthreads();
    
    S[(tid + 1) * B - 1] = sharedAux[tid];  // Improve access pattern
}