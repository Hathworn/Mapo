#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void k4(int *Aux, int *S) {
    // Early exit for block index 0
    if (blockIdx.x == 0) return;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Add preceding block's Aux value to S[tid]
    S[tid] += Aux[blockIdx.x - 1];
}