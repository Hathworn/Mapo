#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Avoid concurrent writes by using atomic operation
    atomicExch(accuracy, *accuracy / N);
}