#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * blockDim.x + threadIdx.x;  // Use blockDim.x for flexibility

    if (tx < numCases) {
        const int label = __float2int_rd(labels[tx]);  // Faster integer conversion
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Direct computation of correctProbs
        int numMax = 0;
        float equalMax = 0.0f;

        if (labelp == maxp) {
            for (int i = 0; i < numOut; ++i) {
                const float prob = probs[i * numCases + tx];  // Load once, used twice
                numMax += (prob == maxp);
                equalMax += (prob == maxp);  // Avoid branching in +=
            }
            correctProbs[tx] = 1.0f / equalMax;  // No branch here, computed as zero if false
        } else {
            correctProbs[tx] = 0.0f;  // Direct assignment for branch not taken
        }
    }
}