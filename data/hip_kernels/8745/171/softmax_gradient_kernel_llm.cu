#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // First level reduction using block-stride loop
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Warp-level reduction
    if (blockDim.x >= 64 && idx < 32) reduction_buffer[idx] += reduction_buffer[idx + 32];
    if (blockDim.x >= 32 && idx < 16) reduction_buffer[idx] += reduction_buffer[idx + 16];
    if (blockDim.x >= 16 && idx < 8) reduction_buffer[idx] += reduction_buffer[idx + 8];
    if (blockDim.x >= 8 && idx < 4) reduction_buffer[idx] += reduction_buffer[idx + 4];
    if (blockDim.x >= 4 && idx < 2) reduction_buffer[idx] += reduction_buffer[idx + 2];
    if (blockDim.x >= 2 && idx < 1) reduction_buffer[idx] += reduction_buffer[idx + 1];

    // Compute gradients after reduction
    if (idx == 0) tmp = reduction_buffer[0];
    __syncthreads();

    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}