#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized device function to compute the inverse translation index
__device__ __forceinline__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int w = ii % d3;
    ii /= d3;
    int z = ii % d2;
    ii /= d2;
    int y = ii % d1;
    ii /= d1;
    int x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

// Optimized device function to compute the translation index
__device__ __forceinline__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int w = ii % d3;
    ii /= d3;
    int z = ii % d2;
    ii /= d2;
    int y = ii % d1;
    ii /= d1;
    int x = ii;
    w /= scale_factor;
    z /= scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x;
    ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
    if (ii >= no_elements) return;

    // Unroll the loops for performance improvement
#pragma unroll
    for (int i = 0; i < scale_factor; i++) {
#pragma unroll
        for (int j = 0; j < scale_factor; j++) {
            int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
            gradInput_data[ii] += gradOutput_data[ipidx];
        }
    }
}