#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;  // Use register for partial sum

    if (eidx < numElements) {
        for (uint i = eidx; i < numElements; i += gridDim.x * DP_BLOCKSIZE) {
            sum += a[i] * b[i];
        }
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();

    for (uint stride = DP_BLOCKSIZE / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }
    
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];  // Reduce only once at the end
    }
}