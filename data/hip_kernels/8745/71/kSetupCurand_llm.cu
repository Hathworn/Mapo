#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Use warp shuffle to initialize multiple states per thread for better performance
    unsigned int lane = tidx % warpSize;
    unsigned int seq = tidx / warpSize;
    hiprand_init(seed, seq, lane, &state[tidx]);
}