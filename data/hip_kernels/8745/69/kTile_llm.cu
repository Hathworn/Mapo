#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Total number of threads
    const int numThreads = blockDim.x * gridDim.x;

    // Loop over tiles, calculate coordinates, and copy
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Directly map source to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}