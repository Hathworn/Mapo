#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DATA float
#define BOOL int
#define MAX_ERR (float)1e-5
#define MAX_EPOCHS 3

#define MAX(a,b) ((a)>(b)?(a):(b))
#define MIN(a,b) ((a)<(b)?(a):(b))

#define OPTIMUM_BLOCK_NUM 4
#define BLOCK_SIDE 16

#define OPTIMUM_BLOCK_NUM_FIRST_LAYER 2
#define BLOCK_SIDE_FIRST_LAYER 32

typedef struct grid_settings {
    int grid[3];
    int block[3];
} grid_settings;

grid_settings gs = { { OPTIMUM_BLOCK_NUM_FIRST_LAYER, OPTIMUM_BLOCK_NUM, OPTIMUM_BLOCK_NUM }, { BLOCK_SIDE_FIRST_LAYER, BLOCK_SIDE, BLOCK_SIDE } };

#define NEURO_INPUT 784
#define NEURO_H_0 56
#define NEURO_H_1 28
#define NEURO_OUTPUT 10
#define TOTAL_PATT 60000
#define NUM_HIDDEN 2
#define TOTAL_LAYER 4

#define NSTREAMS 3

texture<DATA, 2, hipReadModeElementType> texreference_target;

__constant__ DATA alpha_const[1];
__constant__ DATA eta_const[1];

__global__ void updateWeightBiasMat(DATA *delta_weightbias, DATA *weight, int rows, int cols) {
    // Calculate flattened thread index for better memory coalescing
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int thread_index = idy * cols + idx;

    // Ensure threads are within bounds
    if (idx < cols && idy < rows) {
        // Access derivative directly using flattened index
        DATA derivative = delta_weightbias[thread_index];
        // Perform atomic addition to handle concurrent updates safely
        atomicAdd(&weight[thread_index], derivative);
        // Update delta_weightbias with constant alpha factor
        delta_weightbias[thread_index] *= alpha_const[0];
    }
}