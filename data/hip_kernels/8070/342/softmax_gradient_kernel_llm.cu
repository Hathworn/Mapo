#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim; // Move the pointers to the softmax part for this block
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x; // Thread index within the block
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS]; // Shared memory for the reduction
    float tmp = 0.0f;

    // First reduction step - compute dY * Y and sum within each thread
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Reduce the sums within the block into reduction_buffer[0]
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }

    // tmp now holds the sum
    tmp = reduction_buffer[0];

    // Compute the gradient
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}