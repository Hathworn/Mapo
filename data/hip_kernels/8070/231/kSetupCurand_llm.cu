#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x; // Simplified thread index calculation
    hiprand_init(seed, tidx, 0, &state[tidx]); // Initialize hiprand state
}