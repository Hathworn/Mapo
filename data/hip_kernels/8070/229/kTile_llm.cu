#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate the global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Total number of threads
    const int numThreads = blockDim.x * gridDim.x;

    // Loop over all elements assigned to this thread
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;  // Compute target y coordinate
        const uint x = i % tgtWidth;  // Compute target x coordinate
        const uint srcY = y % srcHeight;  // Compute source y coordinate with tiling
        const uint srcX = x % srcWidth;  // Compute source x coordinate with tiling
        tgt[i] = src[srcY * srcWidth + srcX];  // Set target pixel value
    }
}