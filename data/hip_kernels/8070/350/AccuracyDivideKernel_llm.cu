#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread 0 to perform the division for less thread divergence
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}