#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_mean_kernel(float *weights, int n, int size, float *binary, float *mean_arr_gpu)
{
    // Calculate global index for current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread operates within bounds
    if (i < n * size) {
        int f = i / size;

        // Retrieve mean once per feature
        float mean = mean_arr_gpu[f];

        // Assign binary result based on weight sign
        binary[i] = (weights[i] > 0) ? mean : -mean;
    }
}