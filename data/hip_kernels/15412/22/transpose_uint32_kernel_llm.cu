#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_uint32_kernel(uint32_t *src, uint32_t *dst, int src_h, int src_w, int src_align, int dst_align)
{
    // Calculate global index based on block and thread IDs
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate row and column from 1D index
    int i = index % src_h;
    int j = index / src_h;

    // Ensure that the column is within bounds
    if (j < src_w)
    {
        // Perform the transposition while adhering to alignment
        dst[j * (dst_align / 32) + i] = src[i * src_align + j];
    }
}