#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Use block-level stride to ensure all elements are processed
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculation outside of loop for efficiency

    for (int idx = i; idx < n; idx += stride) {
        float diff = truth[idx] - pred[idx];
        error[idx] = diff * diff; // Compute squared difference
        delta[idx] = diff;        // Compute difference (delta)
    }
}