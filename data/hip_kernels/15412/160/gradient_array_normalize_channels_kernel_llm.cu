#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gradient_array_normalize_channels_kernel(float *x, int size, int batch, int channels, int wh_step, float *delta_gpu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    int wh_i = i % wh_step;
    int b = i / wh_step;

    if (i < size) {
        // Optimized loop by avoiding extra computation within loop
        for (int k = 0; k < channels; ++k) {
            const int index = wh_i + k * wh_step + b * wh_step * channels;
            float grad = x[index];
            if (grad > 0) { // Check condition outside delta computation
                float delta = delta_gpu[index];
                delta_gpu[index] = delta * grad; // Direct multiplications to reduce operations
            }
        }
    }
}