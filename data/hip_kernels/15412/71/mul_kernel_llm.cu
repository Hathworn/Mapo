#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a grid-stride loop for better parallel efficiency
    for(; i < N; i += blockDim.x * gridDim.x)
    {
        Y[i * INCY] *= X[i * INCX];
    }
}