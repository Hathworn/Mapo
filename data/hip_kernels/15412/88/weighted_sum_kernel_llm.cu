#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate global thread index using block and thread indices
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use more threads by adjusting block dimension if needed
    if (i < n) {
        // Perform the calculation directly using ternary for compact expression
        c[i] = s[i] * a[i] + (1 - s[i]) * ((b != NULL) ? b[i] : 0.0f);
    }
}