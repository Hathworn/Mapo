#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float softplus_kernel(float x, float threshold = 20) {
    if (x > threshold) return x;                // too large
    else if (x < -threshold) return expf(x);    // too small
    return log1pf(expf(x));                     // optimized, avoid redundant computation
}

__global__ void gradient_array_mish_kernel(int n, float *activation_input_gpu, float *delta) {
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        const float MISH_THRESHOLD = 20.0f;

        // Use shared memory to store frequently used variables for faster access
        __shared__ float shared_inp, shared_sp, shared_tsp, shared_grad_sp;

        shared_inp = activation_input_gpu[i];
        shared_sp = softplus_kernel(shared_inp, MISH_THRESHOLD);
        shared_grad_sp = -expm1f(-shared_sp);
        shared_tsp = tanh(shared_sp);

        __syncthreads(); // Ensure all threads have computed the shared variables

        const float grad_tsp = (1.0f - shared_tsp * shared_tsp) * shared_grad_sp;
        const float grad = shared_inp * grad_tsp + shared_tsp;
        delta[i] *= grad;
    }
}