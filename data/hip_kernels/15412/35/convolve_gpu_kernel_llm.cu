#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolve_gpu_kernel(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n, int size, int pad)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int x = index % in_w;
    int index2 = index / in_w;
    int y = index2 % in_h;
    int fil = index2 / in_h;

    if (fil < n)
    {
        // Precompute output index
        int const output_index = fil * in_w * in_h + y * in_w + x;
        float sum = 0;

        // Loop over channels
        for (int chan = 0; chan < in_c; ++chan)
        {
            // Precompute indices
            int const weights_pre_index = fil * in_c * size * size + chan * size * size;
            int const input_pre_index = chan * in_w * in_h;

            // Loop over filter height and width
            for (int f_y = 0; f_y < size; ++f_y)
            {
                int input_y = y + f_y - pad;
                for (int f_x = 0; f_x < size; ++f_x)
                {
                    int input_x = x + f_x - pad;
                    // Skip out-of-bounds indices
                    if (input_y < 0 || input_x < 0 || input_y >= in_h || input_x >= in_w) continue;

                    // Compute input and weights indices
                    int input_index = input_pre_index + input_y * in_w + input_x;
                    int weights_index = weights_pre_index + f_y * size + f_x;

                    // Accumulate convolution sum
                    sum += input[input_index] * weights[weights_index];
                }
            }
        }

        // Store result in output
        output[output_index] = sum;
    }
}