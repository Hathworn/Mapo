#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Flattened index calculation; simplified grid access
    if (i < n) {
        float p = pred[i];
        float t = truth[i];
        error[i] = (t > 0.0f) ? -__logf(p) : 0.0f; // Use fast math function __logf for better performance
        delta[i] = t - p;
    }
}