#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < N) {
        int spatial_idx = index % spatial;
        int layer_idx = (index / spatial) % layers;
        int batch_idx = index / (spatial * layers);

        // Flatten indices calculation
        int i1 = batch_idx * layers * spatial + layer_idx * spatial + spatial_idx;
        int i2 = batch_idx * layers * spatial + spatial_idx * layers + layer_idx;

        // Conditional copying based on 'forward' flag
        out[forward ? i2 : i1] = x[forward ? i1 : i2];
    }
}