#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate linear index
    int gridStride = blockDim.x * gridDim.x; // Determine grid stride

    while(i < N) {
        Y[OFFY+i*INCY] += ALPHA*X[OFFX+i*INCX]; // Perform axpy operation
        i += gridStride; // Advance to next index in stride
    }
}