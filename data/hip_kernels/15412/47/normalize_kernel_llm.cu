#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;
    
    // Pre-calculate index related variables to avoid repeated computation
    const int f = (index / spatial) % filters;
    const float mean_val = mean[f];
    const float variance_val = variance[f];
    const float epsilon = 0.00001f;

    // Use a temporary variable for computation
    const float normalized = (x[index] - mean_val) / sqrtf(variance_val + epsilon);
    x[index] = normalized;
}