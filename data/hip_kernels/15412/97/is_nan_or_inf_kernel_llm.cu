#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void is_nan_or_inf_kernel(float *input, size_t size, int *pinned_return)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        float val = input[index];
        // Use atomic operation to prevent race condition
        if (isnan(val) || isinf(val))
            atomicMax(pinned_return, 1);
    }
}