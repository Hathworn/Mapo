#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_uint32_kernel_2(uint32_t *src, uint32_t *dst, int src_h, int src_w, int src_align, int dst_align)
{
    __shared__ uint32_t tmp[33 * 32];  // Shared memory for transposing

    int local_x = threadIdx.x % 32;    // Local x within a warp
    int local_y = threadIdx.x / 32;    // Local y within a warp
    int global_index = blockIdx.x;
    int global_x_index = global_index % (src_w / 32);
    int global_y_index = global_index / (src_w / 32);

    int global_x = global_x_index * 32 + local_x;
    int global_y = global_y_index * 32 + local_y;

    uint32_t val = 0;
    if (global_x < src_w && global_y < src_h) {
        val = src[global_y * src_align + global_x];
    }
    tmp[local_x * 33 + local_y] = val;  // Efficient shared memory access
    __syncthreads();
    
    val = tmp[local_y * 33 + local_x];  // Access transposed data
    
    int new_global_x = global_y_index * 32 + local_x;
    int new_global_y = global_x_index * 32 + local_y;

    if (new_global_x < src_h && new_global_y < src_w) {
        dst[new_global_y * (dst_align / 32) + new_global_x] = val;
    }
}