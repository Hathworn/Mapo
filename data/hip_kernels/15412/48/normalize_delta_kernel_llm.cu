#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = index; i < N; i += stride) { // Use loop to handle larger grids
        int f = (i / spatial) % filters;
        
        float inv_sqrt_var = 1.F / (sqrtf(variance[f]) + .000001f); // Precompute invariant parts
        float variance_factor = variance_delta[f] * 2.F / (spatial * batch);
        float mean_factor = mean_delta[f] / (spatial * batch);

        delta[i] = delta[i] * inv_sqrt_var + variance_factor * (x[i] - mean[f]) + mean_factor;
    }
}