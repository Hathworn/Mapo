#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void col2im_gpu_kernel(const int n, const float* data_col, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_im) {
    // Precompute indices outside the loop
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = blockDim.x * gridDim.x;

    for (; index < n; index += gridStride) {
        float val = 0;
        
        // Calculate padded coordinates
        int w = index % width + pad;
        int h = (index / width) % height + pad;
        int c = index / (width * height);
        
        // Precompute common expressions
        int w_stride = w / stride;
        int h_stride = h / stride;
        
        // Calculate start and end positions
        int w_col_start = max((w < ksize) ? 0 : (w - ksize) / stride + 1, 0);
        int w_col_end = min(w_stride + 1, width_col);
        int h_col_start = max((h < ksize) ? 0 : (h - ksize) / stride + 1, 0);
        int h_col_end = min(h_stride + 1, height_col);
        
        // Calculate offsets and coefficients
        int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col;
        int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
        int coeff_w_col = (1 - stride * height_col * width_col);

        // Inner loop unrolling to enhance parallelism
        for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
            int h_offset = h_col * coeff_h_col;
            
            for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
                int w_offset = w_col * coeff_w_col;
                val += data_col[offset + h_offset + w_offset];
            }
        }
        
        // Increment output value
        data_im[index] += val;
    }
}