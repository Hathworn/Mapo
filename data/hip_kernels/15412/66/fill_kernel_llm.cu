#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use loop to handle multiple elements per thread
    for (; index < N; index += stride) {
        X[index * INCX] = ALPHA;
    }
}