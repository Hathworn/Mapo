#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void repack_input_kernel(float *input, float *re_packed_input, int w, int h, int c)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int items_per_channel = w * h;

    // Precompute loop-invariant indices outside the loop
    int total_items = items_per_channel * c;
    int c_pack = index % 32;
    int chan_index = index / 32;
    int chan = (chan_index * 32) % c;
    int i = (chan_index * 32) / c;

    // Combined the loops into one; Conditional check to ensure valid index
    if (index < total_items)
    {
        float src = input[(chan + c_pack) * items_per_channel + i];
        re_packed_input[chan * items_per_channel + i * 32 + c_pack] = src;
    }
}