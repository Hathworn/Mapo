#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fix_nan_and_inf_kernel(float *input, size_t size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if index is within the bounds and process data
    if (index < size) {
        float val = input[index];
        
        // Use ternary operator for cleaner conditional
        input[index] = (isnan(val) || isinf(val)) ? 1.0f / (fabsf((float)index) + 1) : val;
    }
}