#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    // Loop to find the largest element for numerical stability
    for(i = 0; i < n; ++i){
        float val = input[i];
        largest = (val > largest) ? val : largest;
    }
    // Compute exponentials and sum
    for(i = 0; i < n; ++i){
        float e = exp(input[i] / temp - largest / temp);
        sum += e;
        output[i] = e;
    }
    // Normalize the output
    for(i = 0; i < n; ++i){
        output[i] /= sum;
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output)
{
    // Calculate unique index for each thread
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if(b >= batch) return;
    softmax_device(n, input + b * offset, temp, output + b * offset);
}