#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if(i < n) {
        float dc_val = dc[i];
        
        if(da) 
            da[i] += dc_val * s[i];
            
        db[i] += dc_val * (1.0f - s[i]);
        ds[i] += dc_val * (a[i] - b[i]);
    }
}
```
