#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_of_mults_kernel(float *a1, float *a2, float *b1, float *b2, size_t size, float *dst)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        // Simplified expression by using temporary variables
        float temp1 = a1[index] * a2[index];
        float temp2 = b1[index] * b2[index];
        dst[index] = temp1 + temp2;
    }
}