#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
 
    float mean = 0.0f;
    
    // Unrolling loop to improve performance
    for (int i = 0; i < size; i += 4) {
        mean += fabs(weights[f * size + i]);
        if (i + 1 < size) mean += fabs(weights[f * size + i + 1]);
        if (i + 2 < size) mean += fabs(weights[f * size + i + 2]);
        if (i + 3 < size) mean += fabs(weights[f * size + i + 3]);
    }
    mean /= size;

    // Use shared memory for improved cache efficiency
    extern __shared__ float shared_binary[];

    for (int i = 0; i < size; ++i) {
        shared_binary[threadIdx.x * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    }

    // Copy shared memory to global memory
    for (int i = 0; i < size; ++i) {
        binary[f * size + i] = shared_binary[threadIdx.x * size + i];
    }
}