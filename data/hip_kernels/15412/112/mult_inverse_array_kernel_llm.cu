#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_inverse_array_kernel(const float *src_gpu, float *dst_gpu, int size, const float eps)
{
    // Calculate the index based on block and thread dimensions
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index < size) {
        float val = src_gpu[index];
        float sign = (val < 0) ? -1 : 1;
        
        // Use a more efficient way to compute absolute value and powf in single step
        dst_gpu[index] = __powf(fabs(val), eps) * sign;  // Optimize powf calculation
    }
}