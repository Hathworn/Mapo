#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    // Calculate global thread index
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Precompute reused values to reduce operations
    int wh_stride = w * stride;
    int hc = h * c;

    int out_w = i % wh_stride;
    int out_h = (i / wh_stride) % (h * stride);
    int out_c = (i / (wh_stride * h * stride)) % c;
    int b = i / (wh_stride * hc);
    
    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_c = out_c;

    int in_index = b * w * h * c + in_c * w * h + in_h * w + in_w;

    // Use ternary operator for clarity
    float result = scale * (forward ? x[in_index] : out[out_index]);
    if (forward) {
        out[out_index] += result;  // Calculate forward output
    } else {
        atomicAdd(x + in_index, result);  // Update input using atomicAdd
    }
}