#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = blockIdx.x * blockDim.x + threadIdx.x;  // Combine blockIdx.x and threadIdx.x to utilize warp execution
    if (s >= size) return;

    float mean = 0.0f;
    for (int i = 0; i < n; ++i) {
        mean += fabs(input[i * size + s]);
    }
    mean /= n;  // Pre-compute mean outside the inner loop

    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}