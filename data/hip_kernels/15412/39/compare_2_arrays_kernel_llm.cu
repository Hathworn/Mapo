#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compare_2_arrays_kernel(float *one, float *two, int size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= size) return;

    const float val_one = one[index];
    const float val_two = two[index];
    const float diff = 100 * fabs(val_one - val_two) / fabs(val_one);

    // Use conditional to minimize unnecessary printf calls.
    if (diff > 10) 
    {
        printf(" i: %d - one = %f, two = %f, diff = %f %% \n", index, val_one, val_two, diff);
    }
}