#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (i >= N) return;

    // Compute indices
    int in_index = i;
    int in_w = i % w;
    i /= w;
    int in_h = i % h;
    i /= h;
    int in_c = i % c;
    i /= c;
    int b = i % batch;

    // Output channel and offset calculations
    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;

    // Calculate output index
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    // Conditional assignment based on forward flag
    if (forward) {
        out[out_index] = x[in_index];
    } else {
        out[in_index] = x[out_index];
    }
}