#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_scale_bias_kernel(int N, float *x, float *mean, float *variance, float *scales, float *biases, int batch, int filters, int spatial, int inverse_variance, float epsilon)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    const int f = (index / spatial) % filters;

    // Use register to store intermediate results
    float var_adjusted = inverse_variance ? variance[f] : sqrtf(variance[f] + epsilon);
    float val = (x[index] - mean[f]) / var_adjusted;
    val *= scales[f];
    val += biases[f];

    // Avoid any branching for checking NaN and Inf
    if (val == val && abs(val) != INFINITY)
        x[index] = val;
}