#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Compute unique thread index
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    
    // Return early if index exceeds the number of filters
    if (i >= filters) return;

    // Use shared memory to reduce repeated memory access
    extern __shared__ float shared_mem[];
    float *shared_mean = shared_mem;
    float *shared_variance = shared_mem + filters;

    // Load mean and variance into shared memory
    if (threadIdx.x < filters) {
        shared_mean[threadIdx.x] = mean[threadIdx.x];
        shared_variance[threadIdx.x] = variance[threadIdx.x];
    }

    // Sync threads to ensure shared memory is populated
    __syncthreads();
    
    // Initialize variance delta to zero
    float temp = 0.0f;
    
    // Compute variance delta for each batch and spatial dimension
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            temp += delta[index] * (x[index] - shared_mean[i]);
        }
    }
    
    // Scale variance delta with precomputed factor using shared variance
    variance_delta[i] = temp * (-0.5f * powf(shared_variance[i] + .000001f, -1.5f));
}
```
