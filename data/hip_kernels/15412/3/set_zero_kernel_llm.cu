#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_zero_kernel(float *src, int size)
{
    // Calculate grid stride to iterate over elements in a coalesced manner
    int stride = blockDim.x * gridDim.x;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += stride)
    {
        src[i] = 0;
    }
}