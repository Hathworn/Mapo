#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index using 1D block and grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop to handle cases where INCX != 1 and utilize entire grid efficiently
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        X[idx * INCX] *= ALPHA;
    }
}