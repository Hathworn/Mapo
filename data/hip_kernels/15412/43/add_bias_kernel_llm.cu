#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int filters, int spatial, int current_size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < current_size) {
        // Calculate filter index only if needed
        int f = (index / spatial) % filters;
        output[index] += biases[f];
    }
}