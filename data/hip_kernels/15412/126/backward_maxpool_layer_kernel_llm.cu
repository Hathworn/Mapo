#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel function
__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride_x, int stride_y, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + pad - size) / stride_y + 1;
    int w = (in_w + pad - size) / stride_x + 1;
    int c = in_c;
    int area_x = (size - 1) / stride_x;
    int area_y = (size - 1) / stride_y;

    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified calculation for thread index
    if (id >= n) return;

    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    float d = 0;
    for (int l = -area_y; l <= area_y; ++l) {
        for (int m = -area_x; m <= area_x; ++m) {
            int out_w = (j - w_offset) / stride_x + m;
            int out_h = (i - h_offset) / stride_y + l;

            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h) {  // Merged condition check
                int out_index = out_w + w * (out_h + h * (k + c * b));
                if (indexes[out_index] == (i * in_w + j)) {  // Correct index comparison
                    d += delta[out_index];
                }
            }
        }
    }
    prev_delta[i * in_w + j + in_w * in_h * (k + c * b)] += d;  // Direct index access for prev_delta
}