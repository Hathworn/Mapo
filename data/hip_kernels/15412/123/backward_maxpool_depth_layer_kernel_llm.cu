#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_depth_layer_kernel(int n, int w, int h, int c, int batch, float *delta, float *prev_delta, int *indexes)
{
    // Calculate the unique thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the current thread is within bounds
    if (id < n) {
        int index = indexes[id];
        atomicAdd(&prev_delta[index], delta[id]); // Use atomicAdd to avoid race conditions
    }
}