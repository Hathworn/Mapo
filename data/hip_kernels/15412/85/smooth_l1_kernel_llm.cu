#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x 
            + blockIdx.y * blockDim.x * gridDim.x;
    
    if(i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff); // Use optimized `fabsf` for float
        
        if(abs_val < 1.0f) {
            error[i] = diff * diff;
            delta[i] = diff;
        } else {
            error[i] = 2.0f * abs_val - 1.0f;
            delta[i] = copysignf(1.0f, diff); // Use `copysignf` to determine sign
        }
    }
}