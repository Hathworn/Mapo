#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_add_kernel(int N, float ALPHA, float BETA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for loop unrolling

    for (int idx = i; idx < N; idx += stride) { // Loop unrolling for better parallelism
        X[idx * INCX] = X[idx * INCX] * ALPHA + BETA;
    }
}