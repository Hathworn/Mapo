#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expand_array_kernel(const float *src_gpu, float *dst_gpu, int current_size, int groups) 
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Index out of bounds check
    if (index >= current_size) return; 

    // Unroll loop to optimize memory writes
    #pragma unroll
    for (int i = 0; i < groups; ++i) {
        dst_gpu[index + i * current_size] = src_gpu[index];
    }
}