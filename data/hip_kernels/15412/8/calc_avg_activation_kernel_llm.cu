#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calc_avg_activation_kernel(float *src, float *dst, int size, int channels, int batches)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size * batches) {
        int xy = i % size;
        int b = i / size;
        float sum = 0.0f;

        // Unroll the loop for performance improvement
        #pragma unroll
        for (int c = 0; c < channels; ++c) {
            sum += src[xy + size * (c + channels * b)];
        }
        
        // Calculate the average
        dst[i] = sum / channels;
    }
}