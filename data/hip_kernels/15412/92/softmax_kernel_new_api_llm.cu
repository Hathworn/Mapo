#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256

__device__ void softmax_device_new_api(float *input, int n, float temp, int stride, float *output)
{
    // Parallelized reduction for finding largest value
    float largest = -INFINITY;
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        float val = input[i * stride];
        atomicMax(&largest, val); // Use atomic operation to find max
    }
    __shared__ float max_val;
    if (threadIdx.x == 0) max_val = largest;
    __syncthreads();

    // Calculate exponential values and sum them
    float sum = 0;
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        float e = expf(input[i * stride] / temp - max_val / temp);
        atomicAdd(&sum, e); // Use atomic operation to sum
        output[i * stride] = e;
    }
    __shared__ float sum_val;
    if (threadIdx.x == 0) sum_val = sum;
    __syncthreads();

    // Normalize output
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        output[i * stride] /= sum_val;
    }
}

__global__ void softmax_kernel_new_api(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= batch * groups) return;
    int b = id / groups;
    int g = id % groups;
    softmax_device_new_api(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}