#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int k = id % c;
    int b = id / c;

    int out_index = k + c * b;
    float sum = 0.0f;
    for (int i = 0; i < w * h; ++i) {
        int in_index = i + h * w * (k + b * c);
        sum += input[in_index];
    }
    output[out_index] = sum / (w * h); // optimize: compute average once
}