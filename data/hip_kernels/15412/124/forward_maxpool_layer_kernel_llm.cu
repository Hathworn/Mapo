#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride_x, int stride_y, int size, int pad, float *input, float *output, int *indexes)
{
    // Calculate output dimensions
    int h = (in_h + pad - size) / stride_y + 1;
    int w = (in_w + pad - size) / stride_x + 1;

    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Compute output indices
    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % in_c;
    int b = id / (w * h * in_c);

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    int out_index = j + w * (i + h * (k + in_c * b));
    float max_val = -INFINITY;
    int max_idx = -1;

    // Unroll loops for better performance
    for (int l = 0; l < size; ++l) {
        for (int m = 0; m < size; ++m) {
            int cur_h = h_offset + i * stride_y + l;
            int cur_w = w_offset + j * stride_x + m;

            // Only consider valid indices
            bool valid = (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w);
            if (valid) {
                int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
                float val = input[index];
                if (val > max_val) {
                    max_val = val;
                    max_idx = index;
                }
            }
        }
    }

    output[out_index] = max_val;
    if (indexes) indexes[out_index] = max_idx;
}