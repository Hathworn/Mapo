#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stretch_sway_flip_weights_kernel(const float *src_weight_gpu, float *weight_deform_gpu, int nweights, int n, int kernel_size, float angle, int reverse)
{
    const int index = blockIdx.x*blockDim.x + threadIdx.x;
    const int kernel_area = kernel_size * kernel_size;
    const int i = index * kernel_area;

    const int stage_step = (nweights / kernel_area) / 8;  // 8 stages
    const int stage_id = index / stage_step;

    if (i < nweights)
    {
        if (stage_id == 0) {
            // Unroll loop to improve memory access pattern
            for (int y = 0; y < kernel_size; y += 2) {
                for (int x = 0; x < kernel_size; ++x) {
                    weight_deform_gpu[x + y*kernel_size + i] = src_weight_gpu[x + y*kernel_size + i];
                    if (y + 1 < kernel_size) {
                        weight_deform_gpu[x + (y+1)*kernel_size + i] = src_weight_gpu[x + (y+1)*kernel_size + i];
                    }
                }
            }
        }
        else if (stage_id == 1 || stage_id == 2 || stage_id == 3 || stage_id == 4)
        {
            float scale = 0.5;
            if (stage_id == 1) scale = 0.65;
            else if (stage_id == 2) scale = 0.8;
            else if (stage_id == 3) scale = 1.2;
            else if (stage_id == 4) scale = 1.4;

            if (reverse) scale = 1 / scale;

            const int x_c = kernel_size / 2;
            const int y_c = kernel_size / 2;

            float dropout_sum = 0;

            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    float x_s = x_c + (x - x_c) / scale;
                    float y_s = y_c + (y - y_c) / scale;

                    int x_0 = floor(x_s);
                    int x_1 = ceil(x_s);
                    if (x_0 == x_1) x_1 = x_0 + 1;
                    int y_0 = floor(y_s);
                    int y_1 = ceil(y_s);
                    if (y_0 == y_1) y_1 = y_0 + 1;

                    float c_x_0 = x_1 - x_s;
                    float c_x_1 = x_s - x_0;
                    float c_y_0 = y_1 - y_s;
                    float c_y_1 = y_s - y_0;

                    float val = 0;
                    // Use a shared helper function to simplify repetitive code
                    auto add_value = [&](int x, int y, float cx, float cy) {
                        if (x >= 0 && x < kernel_size && y >= 0 && y < kernel_size)
                            val += src_weight_gpu[x + y*kernel_size + i] * cx * cy;
                        else
                            dropout_sum += cx * cy;
                    };

                    add_value(x_0, y_0, c_x_0, c_y_0);
                    add_value(x_1, y_0, c_x_1, c_y_0);
                    add_value(x_0, y_1, c_x_0, c_y_1);
                    add_value(x_1, y_1, c_x_1, c_y_1);

                    weight_deform_gpu[x + y*kernel_size + i] = val;
                }
            }

            // Remove the nested loop for applying compensation
            if (scale > 1) {
                float inverse_scale = 1 / scale;
                for (int y = 0; y < kernel_size; ++y) {
                    for (int x = 0; x < kernel_size; ++x) {
                        weight_deform_gpu[x + y*kernel_size + i] *= inverse_scale;
                    }
                }
            }
        }
        else if (stage_id == 5 || stage_id == 6)
        {
            // Precompute angle coefficients
            if (stage_id == 6) angle = -angle;
            if (reverse) angle = -angle;

            const float cos_a = cosf(angle * 3.14159265 / 180);
            const float sin_a = sinf(angle * 3.14159265 / 180);
            const int x_c = kernel_size / 2;
            const int y_c = kernel_size / 2;

            float dropout_sum = 0;

            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    float x_s = x_c + (x - x_c)*cos_a + (y - y_c)*sin_a;
                    float y_s = y_c - (x - x_c)*sin_a + (y - y_c)*cos_a;

                    int x_0 = floor(x_s);
                    int x_1 = ceil(x_s);
                    if (x_0 == x_1) x_1 = x_0 + 1;
                    int y_0 = floor(y_s);
                    int y_1 = ceil(y_s);
                    if (y_0 == y_1) y_1 = y_0 + 1;

                    float c_x_0 = x_1 - x_s;
                    float c_x_1 = x_s - x_0;
                    float c_y_0 = y_1 - y_s;
                    float c_y_1 = y_s - y_0;

                    float val = 0;

                    auto add_value = [&](int x, int y, float cx, float cy) {
                        if (x >= 0 && x < kernel_size && y >= 0 && y < kernel_size)
                            val += src_weight_gpu[x + y*kernel_size + i] * cx * cy;
                        else
                            dropout_sum += cx * cy;
                    };

                    add_value(x_0, y_0, c_x_0, c_y_0);
                    add_value(x_1, y_0, c_x_1, c_y_0);
                    add_value(x_0, y_1, c_x_0, c_y_1);
                    add_value(x_1, y_1, c_x_1, c_y_1);

                    weight_deform_gpu[x + y*kernel_size + i] = val;
                }
            }

            const float coef = (kernel_size*kernel_size) / (kernel_size*kernel_size - dropout_sum);
            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    weight_deform_gpu[x + y*kernel_size + i] *= coef;
                }
            }
        }
        else if (stage_id == 7)
        {
            // Flip using reverse indexing
            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    weight_deform_gpu[(kernel_size - x - 1) + y*kernel_size + i] = src_weight_gpu[x + y*kernel_size + i];
                }
            }
        }
    }
}