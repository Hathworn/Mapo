#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_copy_kernel(int size, float *src, float *dst)
{
    // Use shared memory to optimize global memory access
    __shared__ float cache[256];  // Adjust size as needed

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int cacheIndex = threadIdx.x;
    
    // Load elements from global to shared memory, avoiding out-of-bounds access
    if (index < size)
    {
        cache[cacheIndex] = src[index];
    }
    
    __syncthreads(); // Ensure all loads to shared memory are complete

    // Write elements from shared to global memory
    if (index < size)
    {
        dst[index] = cache[cacheIndex];
    }
}