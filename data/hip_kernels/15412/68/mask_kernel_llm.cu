#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation for 1D block distribution
    int stride = gridDim.x * blockDim.x; // Calculate stride for grid-stride loop

    for (; i < n; i += stride) // Use grid-stride loop for better occupancy
    {
        if (mask[i] == mask_num) // Efficient conditional check
        {
            x[i] = mask_num; // Perform operation if condition is met
        }
    }
}