#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;
    
    // Compute unique indices for each thread
    int b = id / c;
    int k = id % c;
    int out_index = k + c * b;

    // Pre-compute normalization factor
    float norm_factor = 1.0f / (w * h);
    
    int in_offset = h * w * (k + b * c);
    for (int i = 0; i < w * h; ++i) {
        // Use pre-computed offset and normalization factor
        atomicAdd(&in_delta[in_offset + i], out_delta[out_index] * norm_factor);
    }
}