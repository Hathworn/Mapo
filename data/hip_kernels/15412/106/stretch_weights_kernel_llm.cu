#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stretch_weights_kernel(const float *src_weight_gpu, float *weight_deform_gpu, int nweights, int n, int kernel_size, float scale, int reverse)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int kernel_area = kernel_size * kernel_size;
    const int i = index * kernel_area;
    const int stage_step = (nweights / kernel_area) / 4;  // 4 stages
    const int stage_id = index / stage_step;

    if (i < nweights)
    {
        if (stage_id == 0) {
            // Optimize simple copy using a single loop
            for (int idx = 0; idx < kernel_area; ++idx) {
                weight_deform_gpu[idx + i] = src_weight_gpu[idx + i];
            }
        }
        else
        {
            if (stage_id == 1) scale = 0.65;
            else if (stage_id == 2) scale = 0.8;
            else if (stage_id == 3) scale = 1.3;

            if (reverse) scale = 1 / scale;

            const int x_c = kernel_size / 2;
            const int y_c = kernel_size / 2;

            for (int y = 0; y < kernel_size; ++y) {
                for (int x = 0; x < kernel_size; ++x) {
                    float x_s = x_c + (x - x_c) / scale;
                    float y_s = y_c + (y - y_c) / scale;

                    int x_0 = floor(x_s);
                    int x_1 = ceil(x_s);
                    if (x_0 == x_1) x_1 = x_0 + 1;
                    int y_0 = floor(y_s);
                    int y_1 = ceil(y_s);
                    if (y_0 == y_1) y_1 = y_0 + 1;

                    float c_x_0 = x_1 - x_s;
                    float c_x_1 = x_s - x_0;
                    float c_y_0 = y_1 - y_s;
                    float c_y_1 = y_s - y_0;

                    float val = 0.0f;
                    if (x_0 >= 0 && x_0 < kernel_size && y_0 >= 0 && y_0 < kernel_size)
                        val += src_weight_gpu[x_0 + y_0 * kernel_size + i] * c_x_0 * c_y_0;

                    if (x_1 >= 0 && x_1 < kernel_size && y_0 >= 0 && y_0 < kernel_size)
                        val += src_weight_gpu[x_1 + y_0 * kernel_size + i] * c_x_1 * c_y_0;

                    if (x_0 >= 0 && x_0 < kernel_size && y_1 >= 0 && y_1 < kernel_size)
                        val += src_weight_gpu[x_0 + y_1 * kernel_size + i] * c_x_0 * c_y_1;

                    if (x_1 >= 0 && x_1 < kernel_size && y_1 >= 0 && y_1 < kernel_size)
                        val += src_weight_gpu[x_1 + y_1 * kernel_size + i] * c_x_1 * c_y_1;

                    weight_deform_gpu[x + y * kernel_size + i] = val / scale;  // Combined compensation for dropped items
                }
            }
        }
    }
}