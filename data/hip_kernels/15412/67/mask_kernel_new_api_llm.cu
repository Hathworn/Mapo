#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel_new_api(int n, float *x, float mask_num, float *mask, float val)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = blockDim.x * gridDim.x;
    
    // Loop through elements with grid-stride 
    for (; i < n; i += gridStride) {
        if (mask[i] == mask_num) x[i] = val;
    }
}