#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float mean_sum = 0.0f;  // Use local variable to reduce global memory access
    const int filter_spatial = i * spatial;
    
    for (int j = 0; j < batch; ++j) {
        int base_index = j * filters * spatial + filter_spatial;
        for (int k = 0; k < spatial; ++k) {
            mean_sum += delta[base_index + k];
        }
    }
    
    mean_delta[i] = mean_sum * (-1.F/sqrtf(variance[i] + .000001f));
}