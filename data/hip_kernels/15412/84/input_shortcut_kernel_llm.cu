#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void input_shortcut_kernel(float *in, int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out) {
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Return if outside the bounds
    if (id >= size) return;
    
    // Calculate necessary indices
    int i = id % minw;
    int j = (id / minw) % minh;
    int k = (id / (minw * minh)) % minc;
    int b = id / (minw * minh * minc);
    
    // Use calculated indices to compute output and addition indices
    int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
    int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));
    
    // Perform addition and store result
    out[out_index] = in[out_index] + add[add_index];
}