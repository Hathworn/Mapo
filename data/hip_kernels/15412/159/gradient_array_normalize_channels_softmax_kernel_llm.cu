#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gradient_array_normalize_channels_softmax_kernel(float *x, int size, int batch, int channels, int wh_step, float *delta_gpu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute spatial index within the batch
    int wh_i = i % wh_step;
    int b = i / wh_step;

    if (i < size) {
        for (int k = 0; k < channels; ++k) {
            // Compute index for this channel
            const int index = wh_i + k * wh_step + b * wh_step * channels;
            float delta = delta_gpu[index];

            // Compute gradient using softmax derivative
            float grad = x[index] * (1.0f - x[index]);
            delta = delta * grad;

            // Handle numerical issues
            if (isnan(delta) || isinf(delta)) delta = 0;
            delta_gpu[index] = delta;
        }
    }
}