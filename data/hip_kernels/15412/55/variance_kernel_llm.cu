#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1.F/(batch * spatial - 1);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    float var = 0.0f; // Local variable for variance to optimize memory access
    
    for(int j = 0; j < batch; ++j){
        for(int k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            float diff = x[index] - mean[i]; // Compute difference once
            var += diff * diff; // Accumulate variance
        }
    }
    variance[i] = var * scale; // Store the result
}