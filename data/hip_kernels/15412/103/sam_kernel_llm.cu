#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sam_kernel(float *in_w_h_c, int size, int channel_size, float *scales_c, float *out)
{
    // Calculate index once and reuse
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll loop to improve performance
    if (index < size) {
        #pragma unroll
        for (int i = index; i < size; i += gridDim.x * blockDim.x) {
            out[i] = in_w_h_c[i] * scales_c[i];
        }
    }
}