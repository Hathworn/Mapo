#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Calculate unique global index for each thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;
    
    // Loop through all index values that map linearly this thread
    for(; index < batch * n * size; index += total_threads) {
        int f1 = index / n;
        int f2 = index % n;
        if (f2 <= f1) continue;  // Ensure f2 > f1
        
        float sum = 0;
        float norm1 = 0;
        float norm2 = 0;
        
        for(int b = 0; b < batch; ++b){
            for(int i = 0; i < size; ++i){
                int i1 = b * size * n + f1 * size + i;
                int i2 = b * size * n + f2 * size + i;
                sum += output[i1] * output[i2];
                norm1 += output[i1] * output[i1];
                norm2 += output[i2] * output[i2];
            }
        }
        
        norm1 = sqrtf(norm1);
        norm2 = sqrtf(norm2);
        float norm = norm1 * norm2;
        sum = sum / norm;
        
        for(int b = 0; b < batch; ++b){
            for(int i = 0; i < size; ++i){
                int i1 = b * size * n + f1 * size + i;
                int i2 = b * size * n + f2 * size + i;
                delta[i1] += - scale * sum * output[i2] / norm;
                delta[i2] += - scale * sum * output[i1] / norm;
            }
        }
    }
}