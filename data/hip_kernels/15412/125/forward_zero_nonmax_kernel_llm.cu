#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_zero_nonmax_kernel(int n, float *input, float *output)
{
    // Calculate unique thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Iterate over input/output arrays using stride
    for (; id < n; id += stride) {
        if (input[id] != output[id]) 
            output[id] = 0;
    }
}