#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_scales_dropblock_kernel(float *drop_blocks_scale, int block_size_w, int block_size_h, int outputs, int batch)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds to prevent unnecessary computations
    if (index < batch) {
        const float prob = drop_blocks_scale[index] / (float)outputs;
        
        // Apply scaling calculation only if probability is less than 1
        drop_blocks_scale[index] = (prob < 1.0f) ? 1.0f / (1.0f - prob) : 0.0f;
    }
}