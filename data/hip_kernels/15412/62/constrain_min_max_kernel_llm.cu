#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel by using shared memory and reducing max and min operations
__global__ void constrain_min_max_kernel(int N, float MIN, float MAX, float *X, int INCX)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i < N) {
        // Perform min/max operation only if needed
        float value = X[i * INCX];
        value = value < MIN ? MIN : value;
        value = value > MAX ? MAX : value;
        X[i * INCX] = value;
    }
}