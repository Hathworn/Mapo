#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_weight_updates_kernel(int N, float coef, float *weights_gpu, float *weight_updates_gpu) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if within bounds
    if (i < N) {
        const float w = weights_gpu[i];
        const float wu = weight_updates_gpu[i];
        const float wu_sign = copysignf(1.0f, wu); // Use copysign to determine the sign of wu
        const float abs_limit = fabsf(w * coef);

        // Constrain weight updates
        if (fabsf(wu) > abs_limit) {
            weight_updates_gpu[i] = abs_limit * wu_sign;
        }
    }
}