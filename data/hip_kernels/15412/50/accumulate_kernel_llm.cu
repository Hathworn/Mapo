#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    float temp_sum = 0.0f; // Use a local variable for sum to improve performance
    for (int k = 0; k < n; ++k) {
        temp_sum += x[k * groups + i];
    }
    sum[i] = temp_sum;
}