#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inverse_variance_kernel(int size, float *src, float *dst, float epsilon)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        // Use fast math functions; optimize memory access and computation
        float val = __ldg(&src[index]); // Use __ldg for read-only operations
        dst[index] = rsqrtf(val + epsilon); // Use rsqrtf for fast reciprocal square root
    }
}