#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device_new_api(float *input, int n, float temp, int stride, float *output)
{
    int thread_id = threadIdx.x;
    extern __shared__ float shared_data[];
    float* s_max = &shared_data[0];
    float* s_sum = &shared_data[1];

    float largest = -INFINITY;
    for (int i = thread_id; i < n; i += blockDim.x) {
        float val = input[i * stride];
        atomicMax(&largest, val);  // Use atomic to find max in parallel
    }

    if (thread_id == 0) *s_max = largest;
    __syncthreads();

    float max_val = *s_max;
    float sum = 0;
    for (int i = thread_id; i < n; i += blockDim.x) {
        float e = expf(input[i * stride] / temp - max_val / temp);
        sum += e;
        output[i * stride] = e;
    }

    atomicAdd(s_sum, sum);  // Accumulate sums in shared memory

    __syncthreads();

    sum = *s_sum;  // Only read once all reductions are completed
    for (int i = thread_id; i < n; i += blockDim.x) {
        output[i * stride] /= sum;  // Normalize the outputs
    }
}

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    int thread_id = threadIdx.x;
    extern __shared__ float shared_data[];
    float* s_max = &shared_data[0];
    float* s_sum = &shared_data[1];

    float largest = -INFINITY;
    for (int i = thread_id; i < n; i += blockDim.x) {
        float val = input[i];
        atomicMax(&largest, val);  // Use atomic to find max in parallel
    }

    if (thread_id == 0) *s_max = largest;
    __syncthreads();

    float max_val = *s_max;
    float sum = 0;
    for (int i = thread_id; i < n; i += blockDim.x) {
        float e = expf(input[i] / temp - max_val / temp);
        sum += e;
        output[i] = e;
    }

    atomicAdd(s_sum, sum);  // Accumulate sums in shared memory

    __syncthreads();

    sum = *s_sum;  // Only read once all reductions are completed
    for (int i = thread_id; i < n; i += blockDim.x) {
        output[i] /= sum;  // Normalize the outputs
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;

    int s = id % spatial;
    id = id / spatial;
    int g = id % groups;
    int b = id / groups;
    int goff = group_offset[g] * spatial;
    int boff = b * stride;

    softmax_device_new_api(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}