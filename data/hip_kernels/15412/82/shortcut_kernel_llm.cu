#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) return;

    // Use integer division and mod in a single step for better performance.
    int temp, i, j, k, b;

    b = id / (minw * minh * minc);
    temp = id % (minw * minh * minc);

    k = temp / (minw * minh);
    temp = temp % (minw * minh);

    j = temp / minw;
    i = temp % minw;

    // Use precomputed indices for better memory alignment.
    int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
    int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));
    
    atomicAdd(&out[out_index], add[add_index]); // Improved data race handling with atomic operation
}