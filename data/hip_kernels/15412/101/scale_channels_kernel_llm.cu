#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_channels_kernel(float *in_w_h_c, int size, int channel_size, int batch_size, int scale_wh, float *scales_c, float *out)
{
    // Calculate global thread index
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread should proceed with computations
    if (index < size) {
        // Precompute index components to avoid recalculating
        int index_div_batch_size = index / batch_size;
        int index_div_channel_size = index / channel_size;

        if (scale_wh) {
            // Optimized index computation for wh scaling
            int osd_index = index % channel_size + index_div_batch_size * channel_size;
            out[index] = in_w_h_c[index] * scales_c[osd_index];
        } else {
            // Use precomputed division result for channel scaling
            out[index] = in_w_h_c[index] * scales_c[index_div_channel_size];
        }
    }
}