#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride to handle larger datasets
    for (int index = i; index < N; index += blockDim.x * gridDim.x) {
        // Efficient comparison and zeroing out
        if ((X[index * INCX] * X[index * INCX]) < (ALPHA * ALPHA)) {
            X[index * INCX] = 0;
        }
    }
}