#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_and_expand_array_kernel(const float *src_gpu, float *dst_gpu, int current_size, int groups)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < current_size) {
        float val = 0;
        
        // Unroll the reduction loop for better performance.
        for (int i = 0; i < groups; i += 2) {
            val += src_gpu[index + i * current_size];
            if (i + 1 < groups) {
                val += src_gpu[index + (i + 1) * current_size];
            }
        }
        
        // Compute the average
        val /= groups;

        // Use a single loop for the expansion step.
        for (int i = 0; i < groups; ++i) {
            dst_gpu[index + i * current_size] = val;
        }
    }
}