#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D grid and block indexing
    int gridSize = blockDim.x * gridDim.x; // Calculate stride for loop

    for (; i < N; i += gridSize) { // Loop through elements with stride
        Y[i * INCY] = powf(X[i * INCX], ALPHA); // Utilize loop to handle more elements
    }
}