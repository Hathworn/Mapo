#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_sam_kernel(float *in_w_h_c_delta, int size, int channel_size, float *in_scales_c, float *out_from_delta, float *in_from_output, float *out_state_delta)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if index is within bounds
    if (index < size) {
        float delta = in_w_h_c_delta[index];
        float from_output = in_from_output[index];
        float scales = in_scales_c[index];

        // Update delta for state and from output
        out_state_delta[index] += delta * from_output; // l.delta * from  (should be divided by channel_size?)
        out_from_delta[index] += scales * delta; // input * l.delta

        // Remove commented, obsolete operations
    }
}