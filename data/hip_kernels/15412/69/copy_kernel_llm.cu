#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate the unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop with stride to handle larger arrays and improve parallelism
    while (i < N) {
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
        i += blockDim.x * gridDim.x; // Move index by grid-stride for next iteration
    }
}