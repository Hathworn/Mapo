#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize thread load balance
    int stride = blockDim.x * gridDim.x;

    // Use loop with stride for handling large N
    for (; i < N; i += stride) {
        X[i*INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i*INCX]));
    }
}