#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_bias_kernel(float *output, float *scale, int batch, int filters, int spatial, int current_size)
{
    // Calculate global index
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (index < current_size) {
        // Optimize index calculation to access scale
        int f = (index / spatial) % filters;
        output[index] *= scale[f];
    }
}