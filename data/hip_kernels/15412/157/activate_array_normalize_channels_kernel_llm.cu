#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void activate_array_normalize_channels_kernel(float *x, int size, int batch, int channels, int wh_step, float *output_gpu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    const float eps = 0.0001;
    if (i < size) {
        int wh_i = i % wh_step;  // Calculate wh_i outside of the loops
        int b = i / wh_step;     // Calculate b outside of the loops
        
        float sum = eps;
        
        // Use shared memory to reduce global memory accesses
        __shared__ float shared_x[wh_step * channels];
        
        int thread_id = threadIdx.x;
        
        for (int k = thread_id; k < channels; k += blockDim.x) {
            shared_x[wh_i + k * wh_step] = x[wh_i + k * wh_step + b*wh_step*channels];
        }
        
        __syncthreads(); // Synchronize threads to ensure shared memory is populated
        
        // Sum calculation using shared memory
        for (int k = 0; k < channels; ++k) {
            float val = shared_x[wh_i + k * wh_step];
            if (val > 0) sum += val;
        }

        // Normalization process using shared memory
        for (int k = 0; k < channels; ++k) {
            float val = shared_x[wh_i + k * wh_step];
            if (val > 0) val = val / sum;
            else val = 0;
            output_gpu[wh_i + k * wh_step + b*wh_step*channels] = val; // Write result to global memory
        }
    }
}