#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_dropblock_kernel(float *output, int size, int outputs, float *drop_blocks_scale)
{
    // Calculate global thread index
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundary condition
    if (index < size) {
        // Calculate block index
        const int b = __ldg(&index) / outputs;
        
        // Perform the scaling operation
        output[index] *= __ldg(&drop_blocks_scale[b]);
    }
}