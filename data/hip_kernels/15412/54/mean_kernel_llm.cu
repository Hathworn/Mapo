#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Calculate index using 1D grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float sum = 0.0f;
    int total_elements = batch * spatial;
    
    // Accumulate sum across batch and spatial dimensions
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += x[index];
        }
    }
    
    // Write the mean value for filter i
    mean[i] = sum / total_elements;
}