#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized binarize kernel function
__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified calculation for thread index
    if (i < n)  // Adjusted if condition to reduce number of checks
    {
        binary[i] = (x[i] >= 0) ? 1 : -1;
    }
}