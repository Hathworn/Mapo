#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate a unique thread index for 2D grid with 1D block
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    if (i < n) {
        // Perform the multiplication and addition
        c[i] += a[i] * b[i];
    }
}