#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float hard_mish_yashas_grad(float x)
{
    if (x > 0)
        return 1;
    if (x > -2)
        return x + 1;
    return 0;
}

__device__ float hard_mish_yashas(float x)
{
    if (x > 0)
        return x;
    if (x > -2)
        return x * x / 2 + x;
    return 0;
}

__device__ float mish_yashas(float x)
{
    float e = __expf(x);
    if (x <= -18.0f)
        return x * e;

    float n = e * e + 2 * e;
    if (x <= -5.0f)
        return x * __fdividef(n, n + 2);

    return x - 2 * __fdividef(x, n + 2);
}

__global__ void gradient_array_hard_mish_kernel(int n, float *activation_input_gpu, float *delta)
{
    // Use shared memory to improve memory access latency
    extern __shared__ float shared_activation_input[];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int gridStride = blockDim.x * gridDim.x;

    for (int i = idx; i < n; i += gridStride) {
        shared_activation_input[threadIdx.x] = activation_input_gpu[i];
        __syncthreads();

        // Load data from shared memory
        const float x = shared_activation_input[threadIdx.x];
        delta[i] *= hard_mish_yashas_grad(x);
        __syncthreads();
    }
}