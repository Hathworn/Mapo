#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_local_avgpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride_x, int stride_y, int size, int pad, float *input, float *output)
{
    int h = (in_h + pad - size) / stride_y + 1;
    int w = (in_w + pad - size) / stride_x + 1;
    int c = in_c;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int b = id / (c * h * w);
    int k = (id / (h * w)) % c;
    int i = (id / w) % h;
    int j = id % w;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    int out_index = j + w * (i + h * (k + c * b));
    float avg = 0.0f;
    int counter = 0;

    // Unroll loops to increase memory coalescing
    for (int l = 0; l < size; ++l) {
        int cur_h = h_offset + i * stride_y + l;
        if (cur_h >= 0 && cur_h < in_h) {
            for (int m = 0; m < size; ++m) {
                int cur_w = w_offset + j * stride_x + m;
                if (cur_w >= 0 && cur_w < in_w) {
                    int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
                    avg += input[index];
                    ++counter;
                }
            }
        }
    }
    
    output[out_index] = (counter > 0) ? avg / counter : 0.0f;  // Handle edge case
}