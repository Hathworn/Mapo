#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread ID using block and thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid-stride loop to iterate over all elements processed by this kernel
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x)
    {
        X[idx * INCX] = ALPHA;
    }
}