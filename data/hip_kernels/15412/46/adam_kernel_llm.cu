#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation for 1D grid

    if (index < N) { // Process elements in-bounds
        float B1_t = __powf(B1, t); // Avoid recomputation
        float B2_t = __powf(B2, t);

        float mhat = m[index] / (1.f - B1_t);
        float vhat = v[index] / (1.f - B2_t);

        x[index] += rate * mhat / (sqrtf(vhat) + eps); // Use compound assignment
    }
}