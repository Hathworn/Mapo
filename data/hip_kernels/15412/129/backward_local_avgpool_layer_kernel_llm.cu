#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_local_avgpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride_x, int stride_y, int size, int pad, float *delta, float *prev_delta)
{
    int h = (in_h + pad - size) / stride_y + 1;
    int w = (in_w + pad - size) / stride_x + 1;
    int c = in_c;
    int area_x = (size - 1) / stride_x;
    int area_y = (size - 1) / stride_y;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) return; // Ensure thread index within bounds

    int j = index % in_w;
    int i = (index / in_w) % in_h;
    int k = (index / (in_w * in_h)) % in_c;
    int b = index / (in_w * in_h * in_c);

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    int counter = 0;
    float d = 0.0f;

    // Use register memory to improve performance 
    int w_inner = w * in_h;
    int k_c = k + c * b;

    for (int l = -area_y; l <= area_y; ++l) {
        for (int m = -area_x; m <= area_x; ++m) {
            int out_w = (j - w_offset) / stride_x + m;
            int out_h = (i - h_offset) / stride_y + l;
            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h) { // Check valid indices
                int out_index = out_w + w * (out_h + h * k_c);
                counter++;
                d += delta[out_index];
            }
        }
    }
    if (counter > 0) prev_delta[index] += d / counter; // Aggregate results
}