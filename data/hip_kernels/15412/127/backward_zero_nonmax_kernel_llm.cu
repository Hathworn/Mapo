#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_zero_nonmax_kernel(int n, int *indexes, float *prev_delta)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if within bounds
    if (id >= n) return;
    
    // Zero out non-max indices
    if (indexes[id] != id) prev_delta[id] = 0;
}