#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_array_kernel(float *src, int size, float alpha, float *avg)
{
    // Calculate thread ID
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within bounds
    if (i < size) {
        float temp_avg = avg[i] * (1 - alpha) + src[i] * alpha; // Compute updated avg
        src[i] = temp_avg; // Update src with new avg
        avg[i] = temp_avg; // Update avg with new computed value
    }
}