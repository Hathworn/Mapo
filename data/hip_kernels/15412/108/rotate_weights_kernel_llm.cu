#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rotate_weights_kernel(const float *src_weight_gpu, float *weight_deform_gpu, int nweights, int n, int kernel_size, int reverse)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int kernel_area = kernel_size * kernel_size;
    const int i = index * kernel_area;
    const int stage_step = (nweights / kernel_area) / 4;
    const int stage_id = index / stage_step;

    // Early exit if index is out of bounds
    if (i >= nweights) return;

    // Unroll loops for kernel size to reduce loop overhead
    #pragma unroll
    for (int y = 0; y < kernel_size; ++y) {
        #pragma unroll
        for (int x = 0; x < kernel_size; ++x) {
            const int src_i = x + y * kernel_size + i;
            int dst_i;
            if (stage_id == 0) {
                dst_i = x + y * kernel_size + i; // No rotation
            } else if (stage_id == 1) {
                dst_i = (kernel_size - 1 - y) + x * kernel_size + i; // 90 degree
            } else if (stage_id == 2) {
                dst_i = (kernel_size - 1 - x) + (kernel_size - 1 - y) * kernel_size + i; // 180 degree
            } else { // stage_id == 3
                dst_i = y + (kernel_size - 1 - x) * kernel_size + i; // 270 degree
            }
            if (reverse) {
                weight_deform_gpu[src_i] = src_weight_gpu[dst_i];
            } else {
                weight_deform_gpu[dst_i] = src_weight_gpu[src_i];
            }
        }
    }
}