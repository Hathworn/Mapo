#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void assisted_activation2_kernel(float alpha, float *output, float *gt_gpu, float *a_avg_gpu, int size, int channels, int batches)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float beta = 1 - alpha;

    if (i >= batches * size) return; // Avoid out-of-bounds accesses

    int xy = i % size;
    int b = i / size;

    if (b < batches) {
        if (gt_gpu[i] == 0) {
            for (int c = 0; c < channels; ++c) {
                // Improve memory coalescing by accessing output with a stride pattern
                output[xy + size * (c + channels * b)] *= beta;
            }
        }
    }
}