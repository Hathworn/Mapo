#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output) {
    float sum = 0;
    float largest = -INFINITY;

    // Optimize - Find largest value in input
    for (int i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = fmaxf(val, largest); // Use fmaxf for numerical stability
    }

    // Optimize - Compute exponentials and accumulate the sum
    for (int i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp); // Compute exponentials using stable technique
        sum += e;
        output[i * stride] = e; // Store exponential in output
    }

    // Normalize the output
    float inv_sum = 1.0f / sum; // Precompute reciprocal for normalization
    for (int i = 0; i < n; ++i) {
        output[i * stride] *= inv_sum; // Use precomputed reciprocal to normalize
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset) {
    // Compute global thread ID
    int id = blockIdx.y * gridDim.x  * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;

    int s = id % spatial;
    id /= spatial;
    int g = id % groups;
    int b = id / groups;

    int goff = group_offset[g] * spatial; // Group offset
    int boff = b * stride; // Batch offset

    // Call optimized device function
    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}