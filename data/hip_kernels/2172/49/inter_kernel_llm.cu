#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate unique global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Calculate the total number of elements
    int total_elements = (NX + NY) * B;
    
    // Check if the thread index is within bounds
    if(i < total_elements){
        // Determine batch index and local index within the batch
        int b = i / (NX + NY);
        int j = i % (NX + NY);
        
        // Assign output based on the local index j
        OUT[i] = (j < NX) ? X[b*NX + j] : Y[b*NY + j - NX];
    }
}