#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation using 1D grid and blocks
    if(i < n)
    {
        float m = mask[i];  // Cache mask value to reduce global memory access
        if(m == mask_num) 
             x[i] *= scale;
    }
}