#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    // Compute basic index parameters
    int total_wh = w * h;
    int rem_id = id;
    int k = rem_id % c;
    rem_id /= c;
    int b = rem_id;

    // Pre-calculate indices to reduce repeated calculation
    int out_index = k + c * b;
    float delta_part = out_delta[out_index] / total_wh;

    // Loop across the width-height space for this example-channel pair
    int start_index = h * w * (k + b * c);
    for (int i = 0; i < total_wh; ++i) {
        // Directly updating in_delta without recomputing in_index in each iteration
        in_delta[start_index + i] += delta_part;
    }
}