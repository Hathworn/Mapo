#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    // Initialize shared memory with parallel reduction in mind
    float sum = 0.0f;

    // Loop with stride to cover all spatial elements
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            sum += delta[index];
        }
    }

    local[id] = sum;
    __syncthreads();

    // Reduce using parallel reduction
    for (int offset = threads / 2; offset > 0; offset >>= 1) {
        if (id < offset) {
            local[id] += local[id + offset];
        }
        __syncthreads();
    }

    // Compute mean_delta for this filter
    if(id == 0) {
        mean_delta[filter] = local[0] * (-1.f/sqrtf(variance[filter] + .00001f));
    }
}