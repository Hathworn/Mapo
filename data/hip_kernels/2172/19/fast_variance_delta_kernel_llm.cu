#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    int i, j;
    float sum = 0.0f;

    // Unroll loop to improve performance
    for(j = 0; j < batch; ++j) {
        for(i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            sum += delta[index] * (x[index] - mean[filter]);
        }
    }

    local[id] = sum;
    __syncthreads();

    // Use reduction to aggregate values
    if (threads >= 1024 && id < 512) local[id] += local[id + 512];
    __syncthreads();
    if (threads >= 512 && id < 256) local[id] += local[id + 256];
    __syncthreads();
    if (threads >= 256 && id < 128) local[id] += local[id + 128];
    __syncthreads();
    if (threads >= 128 && id < 64) local[id] += local[id + 64];
    __syncthreads();
    
    if (id < 32) {
        volatile float* vlocal = local;
        vlocal[id] += vlocal[id + 32];
        vlocal[id] += vlocal[id + 16];
        vlocal[id] += vlocal[id + 8];
        vlocal[id] += vlocal[id + 4];
        vlocal[id] += vlocal[id + 2];
        vlocal[id] += vlocal[id + 1];
    }

    if(id == 0) {
        variance_delta[filter] = local[0] * (-0.5f * powf(variance[filter] + .00001f, -1.5f));
    }
}