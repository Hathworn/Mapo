#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= N) return;  // Early exit for threads beyond array bounds

    // Compute indices
    int in_s = i % spatial;
    int in_c = (i / spatial) % layers;
    int b = i / (spatial * layers);

    int i1 = b * layers * spatial + in_c * spatial + in_s;  // Input index
    int i2 = b * layers * spatial + in_s * layers + in_c;   // Output index

    // Conditional assignment based on 'forward' flag
    if (forward) out[i2] = x[i1];
    else out[i1] = x[i2];
}