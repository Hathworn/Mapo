#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D block index for simplicity
    if (i >= filters) return;

    float sum_delta = 0; // Initialize local sum
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j*filters*spatial + i*spatial + k;
            sum_delta += delta[index]; // Accumulate in local variable
        }
    }
    mean_delta[i] = sum_delta * (-1.f/sqrtf(variance[i] + .00001f)); // Compute mean_delta
}