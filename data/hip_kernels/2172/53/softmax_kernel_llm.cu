#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    float sum = 0;
    float largest = -INFINITY;

    // Find the largest element in the input
    for(int i = 0; i < n; ++i){
        float val = input[i*stride];
        largest = fmaxf(val, largest);  // Use fmaxf to simplify
    }

    // Calculate exponentials and their sum
    for(int i = 0; i < n; ++i){
        float e = expf((input[i*stride] - largest) / temp);  // Simplified the expression
        sum += e;
        output[i*stride] = e;
    }

    // Normalize the output based on the sum of exponentials
    for(int i = 0; i < n; ++i){
        output[i*stride] /= sum;
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified ID calculation
    if (id >= batch * groups) return;

    // Calculate batch and group indices
    int b = id / groups;
    int g = id % groups;

    // Call the device function for softmax calculation
    softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}