#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Efficiently stride across elements using grid-stride loop
    for (; i < n; i += gridDim.x * blockDim.x) {
        if (mask[i] == mask_num) {
            x[i] = val;
        }
    }
}