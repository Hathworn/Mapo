#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    // Calculate index based on block and thread information
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        // Calculate the filter index directly using modulo operation
        int f = (index / spatial) % filters;
        
        // Perform normalization using precomputed filter index
        x[index] = (x[index] - mean[f]) / sqrtf(variance[f] + .00001f);
    }
}