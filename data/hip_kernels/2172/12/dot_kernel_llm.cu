#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // For iteration across all elements
    for (int id = index; id < batch * n; id += stride) {
        int f1 = id / n;
        int f2 = id % n;
        if (f2 <= f1) continue; // Skip unnecessary computation

        float sum = 0;
        float norm1 = 0;
        float norm2 = 0;
        
        for (int i = 0; i < size; ++i) {
            int i1 = f1 * size + i;
            int i2 = f2 * size + i;
            sum += output[i1] * output[i2];
            norm1 += output[i1] * output[i1];
            norm2 += output[i2] * output[i2];
        }
        
        norm1 = sqrt(norm1);
        norm2 = sqrt(norm2);
        float norm = norm1 * norm2;
        if (norm > 0) sum = sum / norm; // Avoid division by zero

        for (int i = 0; i < size; ++i) {
            int i1 = f1 * size + i;
            int i2 = f2 * size + i;
            float deltaVal = -scale * sum / norm; // Precompute delta contribution
            delta[i1] += deltaVal * output[i2];
            delta[i2] += deltaVal * output[i1];
        }
    }
}