#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Compute global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x; 
    if (idx >= N) return;

    // Pre-compute constants
    int out_c = c / (stride * stride);
    
    // Calculate indices
    int in_index = idx;
    int b = idx / (w * h * c);
    idx %= w * h * c;
    int in_c = idx / (w * h);
    idx %= w * h;
    int in_h = idx / w;
    int in_w = idx % w;

    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    if (forward) {
        out[out_index] = x[in_index];
    } else {
        out[in_index] = x[out_index];
    }
}