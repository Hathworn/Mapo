#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Flatten block and thread IDs into a single ID
    if (id >= n) return;

    int j = id % w;
    int i = (id / w) % h;  
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    int half_size = size / 2;  // Precalculate half of size for offset
    int out_index = j + w * (i + h * (k + c * b));  // Pre-calculate output index

    float sum = 0;  // Accumulator for delta calculation

    for (int l = -half_size; l <= half_size; ++l) {
        for (int m = -half_size; m <= half_size; ++m) {
            int cur_h = i + l;
            int cur_w = j + m;
            if (cur_h >= 0 && cur_h < h && cur_w >= 0 && cur_w < w) {  // Boundary check
                int index = cur_w + w * (cur_h + h * (k + b * c));  // Calculate index for x
                sum += rate * (x[index] - x[out_index]);  // Update sum with valid values
            }
        }
    }
    delta[out_index] += sum;  // Assign accumulated sum to delta
}