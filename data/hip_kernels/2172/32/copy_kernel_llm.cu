#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified the index calculation

    // Optimized memory handling by directly accessing the data
    while (i < N) {
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
        i += blockDim.x * gridDim.x;
    }
}