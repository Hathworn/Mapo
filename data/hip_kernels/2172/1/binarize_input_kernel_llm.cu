#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = blockIdx.x * blockDim.x + threadIdx.x;
    if (s >= size) return;

    // Use shared memory to reduce redundant global memory access
    extern __shared__ float sharedInput[];
    float mean = 0.0f;

    // Load input data into shared memory
    for (int i = 0; i < n; ++i) {
        sharedInput[i * blockDim.x + threadIdx.x] = fabsf(input[i * size + s]);
        __syncthreads();
    }

    // Calculate mean value
    for (int i = 0; i < n; ++i) {
        mean += sharedInput[i * blockDim.x + threadIdx.x];
    }
    mean /= n;
    
    // Binarize input
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}