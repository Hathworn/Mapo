#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Initialize mean accumulation for this filter
    float sum = 0.0f;

    // Accumulate over batches and spatial dimensions
    for(int j = 0; j < batch; ++j){
        for(int k = 0; k < spatial; ++k){
            int index = j * filters * spatial + i * spatial + k;
            sum += x[index];
        }
    }

    // Multiply accumulated mean with scale factor
    float scale = 1.0f / (batch * spatial);
    mean[i] = sum * scale;
}