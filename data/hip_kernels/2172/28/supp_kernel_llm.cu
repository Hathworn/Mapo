#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Iterate through data with stride
    for(int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        if((X[idx * INCX] * X[idx * INCX]) < (ALPHA * ALPHA)) {
            X[idx * INCX] = 0; // Zero out elements less than ALPHA squared
        }
    }
}