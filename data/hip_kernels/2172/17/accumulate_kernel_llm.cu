#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    float local_sum = 0.0f;  // Use a local variable to store the sum
    for (int k = 0; k < n; ++k) {
        local_sum += x[k * groups + i];
    }
    sum[i] = local_sum;
}