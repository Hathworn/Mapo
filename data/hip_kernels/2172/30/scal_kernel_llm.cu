#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;
    
    // Loop stride to cover entire array
    for (; i < N; i += gridSize) {
        X[i * INCX] *= ALPHA;
    }
}