#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation

    // Use shared memory to minimize global memory accesses
    __shared__ float shared_pred[256];  
    __shared__ float shared_truth[256];  

    if (i < n) {
        shared_pred[threadIdx.x] = pred[i];  
        shared_truth[threadIdx.x] = truth[i];  
        __syncthreads();  // Synchronize threads
        
        float t = shared_truth[threadIdx.x];
        float p = shared_pred[threadIdx.x];
        error[i] = (t) ? -log(p) : 0;  // Calculate error
        delta[i] = t - p;  // Calculate delta
    }
}