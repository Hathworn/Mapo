#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified 1D index calculation
    int gridSize = blockDim.x * gridDim.x; // Calculate the stride for the loop

    while(i < N) // Use loop to cover more indices
    {
        Y[i * INCY] = powf(X[i * INCX], ALPHA); // Use powf for float precision
        i += gridSize; // Move to the next set of indices
    }
}