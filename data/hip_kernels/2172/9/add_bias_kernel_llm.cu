#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    int stride = blockDim.x * gridDim.x; // Calculate total stride

    for (; index < n * size * batch; index += stride) { // Strided loop to cover all elements
        int i = index % size;
        int j = (index / size) % n;
        output[index] += biases[j]; // Access directly using calculated index
    }
}