#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Efficient index calculation
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Process only valid indices
    if(i < n) {
        // Optimize arithmetic operation
        float ai = a[i];
        float bi = b[i];
        c[i] += ai * bi;
    }
}