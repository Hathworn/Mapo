#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate global index of the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if(i < n){
        float dc_val = dc[i]; // Cache dc[i] to reduce memory access
        float s_val = s[i];   // Cache s[i] to reduce memory access

        // Optimize conditions by combining them with cached values
        if(da) da[i] += dc_val * s_val;
        if(db) db[i] += dc_val * (1.0f - s_val);

        // Combine cached a[i] and b[i] operations to reduce memory access
        ds[i] += dc_val * (a[i] - b[i]);
    }
}