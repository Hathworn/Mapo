#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void  fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Use strides to avoid redundant calculations
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += powf((x[index] - mean[filter]), 2);
        }
    }

    __syncthreads();

    // Use reduction to sum the local array more efficiently
    if (id < 32) {
        for (int offset = threads / 2; offset > 0; offset /= 2) {
            if (id < offset) {
                local[id] += local[id + offset];
            }
            __syncthreads();
        }
    }

    if(id == 0) {
        variance[filter] = local[0] / (spatial * batch - 1);
    }
}