#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index with a single line for better readability
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use a loop to allow each thread to handle multiple elements, improving performance
    for (int index = i; index < N; index += stride) {
        X[index * INCX] += ALPHA;
    }
}