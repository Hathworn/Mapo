#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index considering potential 3D grid
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Check bounds to avoid redundant computation
    if (i < N) {
        float value = X[i * INCX];
        value = (value > ALPHA) ? ALPHA : value;   // Apply upper bound
        value = (value < -ALPHA) ? -ALPHA : value; // Apply lower bound
        X[i * INCX] = value;
    }
}