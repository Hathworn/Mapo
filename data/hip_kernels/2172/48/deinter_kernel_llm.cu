#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize the calculation of global thread ID
    int stride = gridDim.x * blockDim.x; // Calculate stride for grid-stride loop
    while(i < (NX + NY) * B) { // Use grid-stride loop for better occupancy
        int b = i / (NX + NY);
        int j = i % (NX + NY);
        if (j < NX) {
            if(X) atomicAdd(&X[b * NX + j], OUT[i]); // Use atomicAdd for safer concurrent writes
        } else {
            if(Y) atomicAdd(&Y[b * NY + j - NX], OUT[i]); // Use atomicAdd
        }
        i += stride;
    }
}