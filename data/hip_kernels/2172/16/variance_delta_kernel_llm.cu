#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    
    float temp_variance = -.5f * powf(variance[i] + .00001f, (float)(-3.f/2.f)); // Precompute constant
    float sum = 0.0f; // Use register for intermediate sum
    
    for(int j = 0; j < batch; ++j) {
        for(int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += delta[index] * (x[index] - mean[i]);
        }
    }

    variance_delta[i] = sum * temp_variance; // Final assignment
}