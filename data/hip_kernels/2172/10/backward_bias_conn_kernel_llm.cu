#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) return; // Early exit if out of bounds

    float sum = 0;
    // Accumulate delta over batch
    for (int b = 0; b < batch; ++b) {
        int i = b * n + index;
        sum += delta[i];
    }
    // Update bias
    bias_updates[index] += sum;
}