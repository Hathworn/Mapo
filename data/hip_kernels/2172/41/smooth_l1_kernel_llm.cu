#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread processes within array bounds
    if(i < n){
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);

        // Optimize condition checking using conditional (ternary) operator
        error[i] = (abs_val < 1) ? (diff * diff) : (2 * abs_val - 1);
        delta[i] = (abs_val < 1) ? diff : ((diff > 0) ? 1 : -1);
    }
}