#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Unrolling batch loop for performance
    for(int b = 0; b < batch; ++b) {
        // Unrolling inner loop for performance
        for(int i = 0; i < size; i += BLOCK*2) {
            int index1 = p + i + size*(filter + n*b);
            int index2 = index1 + BLOCK;
            sum += (p+i < size) ? delta[index1] : 0;
            sum += (p+i+BLOCK < size) ? delta[index2] : 0;
        }
    }
    part[p] = sum;
    __syncthreads();

    // Efficient reduction using loop unrolling and warp aggregation
    if (p < 32) {
        for(int i = 32; i < BLOCK; i += 32) {
            part[p] += part[p + i];
        }
    }
    __syncthreads();
    
    if (p == 0) {
        for(int i = 0; i < 32; ++i) {
            bias_updates[filter] += part[i];
        }
    }
}