#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for loop unrolling

    // Loop unroll to fully utilize the GPU
    for (int i = index; i < N; i += stride) {
        int f = (i / spatial) % filters;
        float inv_std_dev = 1.0f / sqrtf(variance[f] + 0.00001f); // Precompute inverse standard deviation

        // Cache repeated computation to minimize latency
        float var_adj = variance_delta[f] * 2.f * (x[i] - mean[f]) / (spatial * batch);
        float mean_adj = mean_delta[f] / (spatial * batch);

        delta[i] = delta[i] * inv_std_dev + var_adj + mean_adj;
    }
}