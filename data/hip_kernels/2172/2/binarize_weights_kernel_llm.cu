#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x; // Simplify the indexing calculation
    if (f >= n) return;

    float mean = 0.0f;
    float *weights_f = weights + f * size; // Access weights only for this specific index
    float *binary_f = binary + f * size;

    for (int i = 0; i < size; ++i) {
        mean += fabsf(weights_f[i]);
    }
    mean /= size; // Compute the mean

    for (int i = 0; i < size; ++i) {
        binary_f[i] = (weights_f[i] > 0) ? mean : -mean;
    }
}