#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    // Calculate indexes for this thread
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int j = id % in_w;
    int i = (id / in_w) % in_h;
    int k = (id / (in_w * in_h)) % in_c;
    int b = id / (in_w * in_h * in_c);

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;
    int area = (size - 1) / stride;

    float d = 0;

    // Use pragma to unroll loops for optimization
    #pragma unroll
    for (int l = -area; l <= area; ++l) {
        #pragma unroll
        for (int m = -area; m <= area; ++m) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;
            int out_index = out_w + w * (out_h + h * (k + in_c * b));
            int valid = (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h);
            d += (valid && indexes[out_index] == id) ? delta[out_index] : 0;
        }
    }

    atomicAdd(&prev_delta[id], d); // Use atomic for thread-safe update
}