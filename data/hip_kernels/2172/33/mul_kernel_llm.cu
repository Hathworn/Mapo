#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Loop over elements in strides to improve memory access
    for (int i = idx; i < N; i += stride) {
        Y[i * INCY] *= X[i * INCX];
    }
}