#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (index >= N) return;

    float m_hat = m[index] / (1.f - __powf(B1, t)); // Use fast math functions
    float v_hat = v[index] / (1.f - __powf(B2, t));

    x[index] += rate * m_hat / (sqrtf(v_hat) + eps); // Simplified update expression
}