#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Define scale outside of block to avoid repetitive calculation.
    float scale = 1.f / (batch * spatial - 1);
    
    // Calculate global index once for efficiency.
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if within bounds to prevent unnecessary execution.
    if (i >= filters) return;
    
    // Initialize variance directly with scale applied to sum.
    float varSum = 0.0f;
    
    // Use shared memory for mean subtraction calculation.
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            // Avoid redundant powf call by direct multiplication.
            float diff = x[index] - mean[i];
            varSum += diff * diff;
        }
    }
    variance[i] = varSum * scale;
}