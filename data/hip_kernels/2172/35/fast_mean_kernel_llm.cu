#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize fast_mean_kernel by reducing divergence and improving memory operations
__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    int batch_filter_offset = filter * spatial;
    local[id] = 0.0f;

    // Use loop unrolling for spatial dimension
    for (int j = 0; j < batch; ++j) {
        int batch_offset = j * spatial * filters;
        for (int i = id; i < spatial; i += threads) {
            int index = batch_offset + batch_filter_offset + i;
            local[id] += x[index]; // Accumulate values
        }
    }

    __syncthreads();

    // Accumulate results using parallel reduction
    if (id < threads / 2) {
        local[id] += local[id + threads / 2];
    }
    __syncthreads();

    if (id < threads / 4) {
        local[id] += local[id + threads / 4];
    }
    __syncthreads();

    if (id < threads / 8) {
        local[id] += local[id + threads / 8];
    }
    __syncthreads();

    if (id < threads / 16) {
        local[id] += local[id + threads / 16];
    }
    __syncthreads();

    // Calculate mean and store in global memory
    if (id == 0) {
        mean[filter] = local[0] / (spatial * batch);
    }
}