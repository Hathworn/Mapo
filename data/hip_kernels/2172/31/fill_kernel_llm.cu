#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Using a single thread block ID calculation for cleaner code
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_stride = gridDim.x * blockDim.x;

    // Using a stride loop for potentially better utilization of resources
    for (; i < N; i += grid_stride) {
        X[i * INCX] = ALPHA;
    }
}