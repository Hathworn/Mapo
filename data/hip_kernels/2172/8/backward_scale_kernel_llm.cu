#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCK 256

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Update reduction loop with stride and unroll optimization
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index] * x_norm[index];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Efficient parallel reduction within the block
    for (int stride = BLOCK / 2; stride > 0; stride /= 2) {
        if (p < stride) {
            part[p] += part[p + stride];
        }
        __syncthreads();
    }

    // Single thread writes the result
    if (p == 0) {
        scale_updates[filter] = part[0];
    }
}