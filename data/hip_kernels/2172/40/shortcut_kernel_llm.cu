#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    // Calculate unique thread identifier
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Early exit if the identifier exceeds the size
    if (id >= size) return;

    // Calculate indices for output and addition matrices
    int i = id % minw;
    int j = (id / minw) % minh;
    int k = (id / (minw * minh)) % minc;
    int b = id / (minw * minh * minc);

    // Calculate position in output and addition arrays
    int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
    int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));

    // Optimize memory access by reducing arithmetic operations
    float out_val = out[out_index];
    float add_val = add[add_index];
    
    // Update output with weighted addition
    out[out_index] = s1 * out_val + s2 * add_val;
}