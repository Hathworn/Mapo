#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x; 

    // Check element within bounds
    if(i < n){
        // Compute difference
        float diff = truth[i] - pred[i];

        // Store absolute error
        error[i] = fabsf(diff);  // Use fast math function

        // Update delta
        delta[i] = copysignf(1.0f, diff);  // Use copysignf for clean conditional assignment
    }
}