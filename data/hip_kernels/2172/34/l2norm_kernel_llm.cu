#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial) {
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    // Calculate batch and spatial indices
    int b = index / spatial;
    int i = index % spatial;

    // Allocate shared memory for sums
    __shared__ float shared_sum[1024]; 

    // Initialize sum
    float sum = 0;

    // Accumulate the squared values in shared memory
    for (int f = threadIdx.x; f < filters; f += blockDim.x) {
        int dataIndex = b * filters * spatial + f * spatial + i;
        sum += powf(x[dataIndex], 2);
    }

    // Perform reduction to find total sum in shared memory
    atomicAdd(&shared_sum[threadIdx.x], sum);

    __syncthreads(); // Synchronize threads

    // Get total sum for normalization
    if (threadIdx.x == 0) {
        sum = 0;
        for (int j = 0; j < blockDim.x; ++j) {
            sum += shared_sum[j];
        }
        sum = sqrtf(sum);
        if(sum == 0) sum = 1;
    }

    __syncthreads(); // Synchronize threads

    // Normalize the values
    for (int f = threadIdx.x; f < filters; f += blockDim.x) {
        int dataIndex = b * filters * spatial + f * spatial + i;
        x[dataIndex] /= sum;
        dx[dataIndex] = (1 - x[dataIndex]) / sum;
    }
}