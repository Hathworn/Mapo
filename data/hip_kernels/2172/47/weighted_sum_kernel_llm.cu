#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify index calculation
    int gridStride = gridDim.x * blockDim.x;        // Define grid stride
    for (; i < n; i += gridStride) {                // Use stride loop for full grid utilization
        float b_val = (b != nullptr) ? b[i] : 0.0f; // Pre-compute b[i] value or zero
        c[i] = s[i] * a[i] + (1.0f - s[i]) * b_val; // Calculate c[i]
    }
}