#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate the thread index to ensure efficient memory access
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if (i < n) {
        float t = truth[i];
        float p = pred[i];
        
        // Pre-compute common values for error calculation to improve performance
        float log_p = logf(p + 1e-7f);
        float log_1_minus_p = logf(1.0f - p + 1e-7f);
        
        error[i] = -t * log_p - (1.0f - t) * log_1_minus_p;
        delta[i] = t - p;
    }
}