#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Use grid-stride loop to handle large problems efficiently
    for (; i < N; i += blockDim.x * gridDim.x) 
    {
        X[i * INCX] = ALPHA;
    }
}