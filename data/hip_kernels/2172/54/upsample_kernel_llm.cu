#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation
    if (index >= N) return;
    
    // Precompute values outside the nested calculations
    int totalStrideW = w * stride;
    int totalStrideH = h * stride;
    int whc = w * h * c;

    int out_index = index;
    int out_w = out_index % totalStrideW;
    int out_h = (out_index / totalStrideW) % totalStrideH;
    int out_c = (out_index / (totalStrideW * totalStrideH)) % c;
    int b = (out_index / (totalStrideW * totalStrideH * c)) % batch;

    // Calculate input coordinates
    int in_w = out_w / stride;
    int in_h = out_h / stride;
    
    // Flatten the input index
    int in_index = b * whc + out_c * w * h + in_h * w + in_w;

    if (forward) 
        out[out_index] += scale * x[in_index];
    else 
        atomicAdd(x + in_index, scale * out[out_index]);
}