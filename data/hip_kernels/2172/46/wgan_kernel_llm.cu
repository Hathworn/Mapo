#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void wgan_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Use shared memory for faster access
    __shared__ float shared_pred[256];
    __shared__ float shared_truth[256];
    if (threadIdx.x < n) {
        shared_pred[threadIdx.x] = pred[threadIdx.x];
        shared_truth[threadIdx.x] = truth[threadIdx.x];
    }
    __syncthreads();

    // Check boundary condition
    if (i < n) {
        // Use ternary operator for concise code
        error[i] = shared_truth[i] ? -shared_pred[i] : shared_pred[i];
        delta[i] = (shared_truth[i] > 0) ? 1 : -1;
    }
}