#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv2genericrev(float *input, float *kernel, float *output, int input_n, int input_h, int input_w, int kernel_n, int kernel_h, int kernel_w, float alpha, int stride_h, int stride_w)
{
    // Calculate output dimensions
    int output_h = input_h - (kernel_h - 1) * stride_h;
    int output_w = input_w - (kernel_w - 1) * stride_w;

    // Output offset for the specific thread block
    int kk = blockIdx.x;
    int ii = blockIdx.y;
    output += (kk * input_n + ii) * output_h * output_w;

    // Setup shared memory
    __shared__ float shared_output[CUDA_SHARED_MEM_SIZE];

    // Simplify thread indexing
    int batch = blockIdx.z;
    int kid = threadIdx.x;
    int tid = threadIdx.y * blockDim.x + kid;
    int nthreads = blockDim.x * blockDim.y;

    // Define shared memory for outputs
    float* output_s = shared_output + tid * output_h * output_w;

    // Convolution computation
    for (int yy = threadIdx.y; yy < output_h; yy += blockDim.y) {
        float* output_p = output_s + yy * output_w;
        for (int xx = 0; xx < output_w; xx++) {
            float* input_p = input + (ii + batch * input_n) * input_h * input_w + yy * stride_h * input_w + xx * stride_w;
            float* kernel_p = kernel + (kk + batch * kernel_n) * kernel_w * kernel_h;
            float sum = 0;
            for (int ky = 0; ky < kernel_h; ky++, input_p += input_w, kernel_p += kernel_w) {
                for (int kx = kid; kx < kernel_w; kx += blockDim.x) {
                    sum += input_p[kx] * kernel_p[kx];
                }
            }
            output_p[xx] = sum;
        }
    }
    __syncthreads();

    // Reduction step within shared memory and write-back
    for (int i = tid; i < output_w * output_h; i += nthreads) {
        for (int k = 1; k < nthreads; k++) {
            shared_output[i] += shared_output[k * output_h * output_w + i];
        }
        output[i] += alpha * shared_output[i];
    }
}