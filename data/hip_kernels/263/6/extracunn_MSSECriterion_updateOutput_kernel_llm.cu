#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void extracunn_MSSECriterion_updateOutput_kernel(float* output, float* input, float* target, int nframe, int dim)
{
    extern __shared__ float buffer[];
    int k = blockIdx.x;
    float* input_k = input + k * dim;
    float* target_k = target + k * dim;

    int i = threadIdx.x;
    float z_partial = 0.0f;

    // Calculate partial sum of z
    while (i < dim)
    {
        float z = input_k[i] - target_k[i];
        z_partial += z;
        i += blockDim.x;
    }

    buffer[threadIdx.x] = z_partial;
    __syncthreads();

    // Reduce all partial sums
    if (threadIdx.x == 0)
    {
        float sum = 0.0f;
        for (int j = 0; j < blockDim.x; j++)
        {
            sum += buffer[j];
        }
        *output = sum * sum / (-2.0f * dim * dim);
    }
}