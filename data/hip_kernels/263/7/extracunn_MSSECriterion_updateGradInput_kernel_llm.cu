#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void extracunn_MSSECriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, float norm, int nframe, int dim)
{
    int k = blockIdx.x;
    float *gradInput_k = gradInput + k * dim;
    float *input_k = input + k * dim;
    float *target_k = target + k * dim;

    __shared__ float buffer[MSSECRITERION_THREADS];

    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;
    float local_sum = 0.0;

    // Compute local sum of differences
    for (int i = i_start; i < i_end; i += i_step)
    {
        float z = input_k[i] - target_k[i];
        local_sum += z;
    }
    buffer[threadIdx.x] = local_sum;
    __syncthreads();

    // Perform reduction in shared memory
    if (threadIdx.x == 0)
    {
        float sum = 0.0;
        for (int i = 0; i < blockDim.x; i++)
        {
            sum += buffer[i];
        }
        buffer[0] = sum; // Store final reduced sum in buffer[0]
    }
    __syncthreads();

    // Scale and write the gradient input
    float final_sum = buffer[0];
    for (int i = i_start; i < i_end; i += i_step)
    {
        gradInput_k[i] = norm * final_sum;
    }
}
```
