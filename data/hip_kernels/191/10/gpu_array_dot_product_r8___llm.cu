#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_array_dot_product_r8__(size_t tsize, const double *arr1, const double *arr2, volatile double *dprod)
{
    extern __shared__ double dprs_r8[]; // Shared memory for partial sums
    size_t l = blockIdx.x * blockDim.x + threadIdx.x;
    double dpr = 0.0;

    // Unroll loop for better performance
    while (l < tsize) {
        dpr += arr1[l] * arr2[l];
        l += gridDim.x * blockDim.x;
    }

    dprs_r8[threadIdx.x] = dpr;
    __syncthreads();

    // Reduce using a more efficient method
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            dprs_r8[threadIdx.x] += dprs_r8[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        // Acquire lock using atomic operation
        while (atomicCAS(&dot_product_wr_lock, 0, 1) != 0) { }
        
        *dprod += dprs_r8[0];
        __threadfence();

        // Release lock
        atomicExch(&dot_product_wr_lock, 0);
    }
}