#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_array_scale_r8__(size_t tsize, double *arr, double val)
{
    // Calculate global thread ID
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll the loop by process multiple elements per thread
    size_t stride = gridDim.x * blockDim.x;
    for (size_t i = idx; i < tsize; i += stride) {
        arr[i] *= val;
    }
}