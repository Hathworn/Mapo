#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_array_dot_product_r4__(size_t tsize, const float *arr1, const float *arr2, volatile float *dprod)
{
    extern __shared__ float dprs_r4[]; // shared memory for partial dot products

    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = gridDim.x * blockDim.x;

    // Initialize partial dot product for this thread
    float dpr = 0.0f;

    // Parallel accumulation of dot product
    for (size_t l = idx; l < tsize; l += stride) {
        dpr += arr1[l] * arr2[l];
    }

    // Store partial result in shared memory
    dprs_r4[threadIdx.x] = dpr;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            dprs_r4[threadIdx.x] += dprs_r4[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Add block's contribution to final result
    if (threadIdx.x == 0) {
        int lock;
        do {
            lock = atomicMax(&dot_product_wr_lock, 1); // Try to acquire lock
        } while (lock == 1);

        *dprod += dprs_r4[0]; // Update global dot product
        __threadfence(); // Ensure the update is visible before releasing lock
        
        atomicExch(&dot_product_wr_lock, 0); // Release lock
    }
}