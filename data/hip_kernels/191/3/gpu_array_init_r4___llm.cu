#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_array_init_r4__(size_t tsize, float *arr, float val)
{
    // Calculate global thread index
    size_t _ti = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride to assign values across entire array
    size_t stride = gridDim.x * blockDim.x;

    for(size_t l = _ti; l < tsize; l += stride) {
        arr[l] = val; // Initialize array with val
    }
}