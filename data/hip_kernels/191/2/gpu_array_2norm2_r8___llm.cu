#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_array_2norm2_r8__(size_t arr_size, const double *arr, double *bnorm2)
/** Computes the squared Euclidean (Frobenius) norm of an array arr(0:arr_size-1)
INPUT:
# arr_size - size of the array;
# arr(0:arr_size-1) - array;
OUTPUT:
# bnorm2[0:gridDim.x-1] - squared 2-norm of a sub-array computed by each CUDA thread block;
**/
{
    size_t i, n;
    double _thread_norm2 = 0.0;
    extern __shared__ double thread_norms2_r8[];

    // Calculate global index and stride
    n = blockDim.x * gridDim.x;
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Compute squared norm for assigned elements
    for (i = index; i < arr_size; i += n) {
        _thread_norm2 += arr[i] * arr[i];
    }
    
    // Store the thread result in shared memory
    thread_norms2_r8[threadIdx.x] = _thread_norm2;
    __syncthreads();

    // Reduce results within the block
    if (threadIdx.x == 0) {
        double block_norm2 = 0.0;
        for (i = 0; i < blockDim.x; i++) {
            block_norm2 += thread_norms2_r8[i];
        }
        bnorm2[blockIdx.x] = block_norm2;
    }
}