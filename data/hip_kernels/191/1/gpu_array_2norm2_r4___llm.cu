#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_array_2norm2_r4__(size_t arr_size, const float *arr, float *bnorm2)
/** Computes the squared Euclidean (Frobenius) norm of an array arr(0:arr_size-1)
INPUT:
# arr_size - size of the array;
# arr(0:arr_size-1) - array;
OUTPUT:
# bnorm2[0:gridDim.x-1] - squared 2-norm of a sub-array computed by each CUDA thread block;
**/
{
    size_t n = gridDim.x * blockDim.x;
    float _thread_norm2 = 0.0f;
    extern __shared__ float thread_norms2_r4[];

    // Accumulate squared values for the assigned portion of the array
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < arr_size; i += n) {
        _thread_norm2 += arr[i] * arr[i];
    }
    thread_norms2_r4[threadIdx.x] = _thread_norm2;
    __syncthreads();

    // Use a single thread to accumulate results and write to output
    if (threadIdx.x == 0) {
        float block_sum = 0.0f;
        for (size_t i = 0; i < blockDim.x; i++) {
            block_sum += thread_norms2_r4[i];
        }
        bnorm2[blockIdx.x] = block_sum;
    }
}