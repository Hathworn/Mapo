#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_array_scale_r4__(size_t tsize, float *arr, float val)
{
    // Calculate the global thread index
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Process elements in stride of the total number of threads
    if (idx < tsize) {
        arr[idx] *= val;
    }
    
    // Stride loop to cover all elements
    size_t gridStride = gridDim.x * blockDim.x;
    for (size_t i = idx + gridStride; i < tsize; i += gridStride) {
        arr[i] *= val;
    }
    return;
}