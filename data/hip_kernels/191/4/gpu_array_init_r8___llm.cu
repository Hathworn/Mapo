#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_array_init_r8__(size_t tsize, double *arr, double val)
{
    // Compute global thread index
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure idx is within array bounds
    if (idx < tsize) {
        arr[idx] = val;
    }
    return;
}