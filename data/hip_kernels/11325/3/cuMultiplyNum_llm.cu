#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define PI 3.141592653589793
#define BLOCKSIZE 1024

__global__ void cuMultiplyNum(float *dst, float num, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread id is within bounds and perform multiplication.
    if (id < size) {
        dst[id] *= num;
    }
}