#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce(float* d_out, float* d_in) {
    extern __shared__ float sdata[];

    int globId = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;

    // Load input into shared memory
    sdata[tid] = globId < blockDim.x * gridDim.x ? d_in[globId] : 0.0f; // Avoid reading out of bounds
    __syncthreads();

    // Perform log step reductions
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}