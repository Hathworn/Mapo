#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Laplace(float* d_out, float* d_in) {
    // Calculate rowID and colID more efficiently
    int colID = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int rowID = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int pos = rowID * (blockDim.x + 2) + colID;

    // Use shared memory to reduce global memory access
    extern __shared__ float shared_in[];
    int shared_pos = threadIdx.y * blockDim.x + threadIdx.x;
    shared_in[shared_pos] = d_in[pos];
    __syncthreads();

    // Improved memory access by using shared memory when possible
    if (threadIdx.x > 0 && threadIdx.x < blockDim.x - 1 && threadIdx.y > 0 && threadIdx.y < blockDim.y - 1) {
        d_out[pos] = (shared_in[shared_pos - 1] + shared_in[shared_pos + 1] +
                      shared_in[shared_pos - blockDim.x] + shared_in[shared_pos + blockDim.x]) / 4.0; 
    } else {
        d_out[pos] = (d_in[pos - 1] + d_in[pos + 1] +
                      d_in[pos - blockDim.x - 2] + d_in[pos + blockDim.x + 2]) / 4.0; // Handle boundary conditions
    }
}