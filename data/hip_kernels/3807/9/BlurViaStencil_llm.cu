#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BlurViaStencil(float* d_out, float* d_in) {
    const float kernel[3][3] = {0.04, 0.12, 0.04,
                                0.12, 0.36, 0.12,
                                0.04, 0.12, 0.04};
    int rowID = blockIdx.x + 1;
    int colID = threadIdx.x + 1;
    int pos = rowID * (blockDim.x + 2) + colID;

    // Compute indices to avoid repeated calculations
    int posRowAbove = pos - blockDim.x - 2;
    int posCurrentRow = pos;
    int posRowBelow = pos + blockDim.x + 2;

    // Apply stencil kernel to input matrix
    d_out[pos] = d_in[posRowAbove - 1] * kernel[0][0]
               + d_in[posRowAbove] * kernel[0][1]
               + d_in[posRowAbove + 1] * kernel[0][2]
               + d_in[posCurrentRow - 1] * kernel[1][0]
               + d_in[posCurrentRow] * kernel[1][1]
               + d_in[posCurrentRow + 1] * kernel[1][2]
               + d_in[posRowBelow - 1] * kernel[2][0]
               + d_in[posRowBelow] * kernel[2][1]
               + d_in[posRowBelow + 1] * kernel[2][2];
}