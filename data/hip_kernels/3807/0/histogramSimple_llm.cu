#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// In CUDA we trust.

// When compiling, use -std=c++11 or higher.

__global__ void histogramSimple(int* d_out, const int* d_in, const int BINS_COUNT) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop over the data to handle more elements per thread for better load balancing 
    for (int i = tid; i < BINS_COUNT; i += stride) {
        atomicAdd(&(d_out[d_in[i] % BINS_COUNT]), 1);
    }
}