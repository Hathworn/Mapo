#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce(float* d_out, float* d_in) { // Parallel summation: steps = O(log(N)), work = O(N * log(N))
    extern __shared__ float sdata[];

    int globId = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;

    // Reduce global memory access: load only if within bounds
    if (globId < gridDim.x * blockDim.x) {
        sdata[tid] = d_in[globId];
    } else {
        sdata[tid] = 0.0f; // Set to zero if out of bounds
    }
    __syncthreads();

    for (int s = blockDim.x >> 1; s > 0; s >>= 1) { // Unrolling loop for better performance
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}