#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogramSimple(int* d_out, const int* d_in, const int BINS_COUNT) {
    // Calculate global thread ID
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int data = d_in[tid];

    // Atomic operation reduces contention in shared memory
    atomicAdd(&(d_out[data % BINS_COUNT]), 1);
}