#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void montecarlo(float* d_out, float __lowx, float __highx, float __lowy, float __highy, int __iters) {
    __shared__ float lowx, highx, lowy, highy;
    __shared__ int iters;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize shared variables only once per block
    if (threadIdx.x == 0) {
        lowx = __lowx;
        highx = __highx;
        lowy = __lowy;
        highy = __highy;
        iters = __iters;
    }
    __syncthreads();

    hiprandState localState;
    hiprand_init(tid, 0, 0, &localState);

    float x, y, tempSum = 0.0f;

    // Use strided loop to reduce thread divergence and improve memory access
    for (int i = tid; i < iters * gridDim.x * blockDim.x; i += gridDim.x * blockDim.x) {
        x = lowx + hiprand_uniform(&localState) * (highx - lowx);
        y = lowy + hiprand_uniform(&localState) * (highy - lowy);
        tempSum += expf(-x * x - y * y);
    }

    d_out[tid] = tempSum;
}