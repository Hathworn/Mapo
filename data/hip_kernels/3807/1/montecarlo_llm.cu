#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void montecarlo(float* d_out, float __lowx, float __highx, float __lowy, float __highy, int __iters) {
    __shared__ float lowx, highx, lowy, highy;
    __shared__ int iters;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize shared variables once per block.
    if (threadIdx.x == 0) {
        lowx = __lowx;
        highx = __highx;
        lowy = __lowy;
        highy = __highy;
        iters = __iters;
    }
    __syncthreads();

    hiprandState localState;
    hiprand_init(tid, 0, 0, &localState);

    // Use registers to store loop variables and reduce accesses to shared memory.
    int i;
    float x, y, tempSum = 0.f;
    float range_x = highx - lowx;
    float range_y = highy - lowy;

    // Each thread calculates its own summation.
    for (i = 0; i < iters; i++) {
        x = lowx + hiprand_uniform(&localState) * range_x;
        y = lowy + hiprand_uniform(&localState) * range_y;
        tempSum += expf(-x * x - y * y); // Use expf for single precision
    }
    d_out[tid] = tempSum;
}