#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to perform prescan
__global__ void prescan(float* d_in, int nGlobe, int step, int upSweep) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = 2 * tid * (step + 1); // Calculate shared offset

    // Calculate indices for from and to
    int from = offset + step;
    int to = offset + 2 * step + 1;

    if (upSweep) {
        d_in[to] += d_in[from]; // Up-sweep: combine results
    } else {
        int temp = d_in[from]; // Use a temporary variable to swap
        d_in[from] = d_in[to];
        d_in[to] += temp; // Down-sweep: add and update result
    }
}