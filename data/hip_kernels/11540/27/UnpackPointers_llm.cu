#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;

    // Optimize loop limits and data access patterns
    const int stride = blockDim.x;

    for (int y = 0; y < h / 32; y++) {
        for (int x = tx; x < w; x += stride) {
            unsigned int val = minmax[y * w + x];

            if (val) {
                #pragma unroll // Unroll the inner loop for better performance
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}