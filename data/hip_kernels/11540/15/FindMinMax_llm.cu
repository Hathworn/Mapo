#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * blockDim.x + tx;  // Calculate global x with blockDim
    const int y = blockIdx.y * blockDim.y;       // Calculate global y with blockDim
    int p = y * pitch + x;
    
    float val = (x < width) ? d_Data[p] : d_Data[p - x];
    minvals[tx] = val;
    maxvals[tx] = val;

    for (int ty = 1; ty < blockDim.y; ty++) {  // Use blockDim.y instead of constant
        p += pitch;
        if (x < width) {
            float val = d_Data[p];
            if (val < minvals[tx])
                minvals[tx] = val;
            if (val > maxvals[tx])
                maxvals[tx] = val;
        }
    }
    __syncthreads();

    for (int d = blockDim.x / 2; d > 0; d >>= 1) {  // Use reduction pattern technique
        if (tx < d) {
            if (minvals[tx + d] < minvals[tx])
                minvals[tx] = minvals[tx + d];
            if (maxvals[tx + d] > maxvals[tx])
                maxvals[tx] = maxvals[tx + d];
        }
        __syncthreads();
    }

    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr + 0] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}