#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Calculate indices using modern indexing method
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;
    
    // Ensure within boundary before performing subtraction
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // __syncthreads() not needed as no shared memory is used
}