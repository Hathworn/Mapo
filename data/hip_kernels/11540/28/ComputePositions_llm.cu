#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputePositions(float *g_Data1, float *g_Data2, float *g_Data3, int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h) {
    int i = __mul24(blockIdx.x, POSBLK_SIZE) + threadIdx.x;
    if (i >= numPts) return;

    int p = d_Ptrs[i];
    
    float val[7];
    // Pre-fetch data to local variables to minimize global memory access
    float g_Data2_p = g_Data2[p];
    float g_Data2_p_minus_1 = g_Data2[p - 1];
    float g_Data2_p_plus_1 = g_Data2[p + 1];
    float g_Data2_p_minus_w = g_Data2[p - w];
    float g_Data2_p_plus_w = g_Data2[p + w];
    float g_Data3_p = g_Data3[p];
    float g_Data1_p = g_Data1[p];

    val[0] = g_Data2_p;
    val[1] = g_Data2_p_minus_1;
    val[2] = g_Data2_p_plus_1;

    float dx = 0.5f * (val[2] - val[1]);
    float dxx = 2.0f * val[0] - val[1] - val[2];
    
    val[3] = g_Data2_p_minus_w;
    val[4] = g_Data2_p_plus_w;

    float dy = 0.5f * (val[4] - val[3]);
    float dyy = 2.0f * val[0] - val[3] - val[4];

    val[5] = g_Data3_p;
    val[6] = g_Data1_p;

    float ds = 0.5f * (val[6] - val[5]);
    float dss = 2.0f * val[0] - val[5] - val[6];

    // Calculate mixed derivatives with reduced repeated access to global memory
    float dxy = 0.25f * (g_Data2[p + w + 1] + g_Data2[p - w - 1] - g_Data2[p - w + 1] - g_Data2[p + w - 1]);
    float dxs = 0.25f * (g_Data3[p + 1] + g_Data1[p - 1] - g_Data1[p + 1] - g_Data3[p - 1]);
    float dys = 0.25f * (g_Data3[p + w] + g_Data1[p - w] - g_Data3[p - w] - g_Data1[p + w]);

    // Compute determinant and inverse matrix components
    float idxx = dyy * dss - dys * dys;
    float idxy = dys * dxs - dxy * dss;
    float idxs = dxy * dys - dyy * dxs;
    float idyy = dxx * dss - dxs * dxs;
    float idys = dxy * dxs - dxx * dys;
    float idss = dxx * dyy - dxy * dxy;
    float det = idxx * dxx + idxy * dxy + idxs * dxs;
    float idet = 1.0f / det;

    // Calculate offsets
    float pdx = idet * (idxx * dx + idxy * dy + idxs * ds);
    float pdy = idet * (idxy * dx + idyy * dy + idys * ds);
    float pds = idet * (idxs * dx + idys * dy + idss * ds);

    // Boundary check for offsets
    if (pdx < -0.5f || pdx > 0.5f || pdy < -0.5f || pdy > 0.5f || pds < -0.5f || pds > 0.5f) {
        pdx = __fdividef(dx, dxx);
        pdy = __fdividef(dy, dyy);
        pds = __fdividef(ds, dss);
    }

    float dval = 0.5f * (dx * pdx + dy * pdy + ds * pds);

    // Store results
    d_Sift[i + 0 * maxPts] = (p % w) + pdx;
    d_Sift[i + 1 * maxPts] = (p / w) + pdy;
    d_Sift[i + 2 * maxPts] = d_ConstantA[0] * exp2f(pds * d_ConstantB[0]);
    d_Sift[i + 3 * maxPts] = val[0] + dval;

    float tra = dxx + dyy;
    det = dxx * dyy - dxy * dxy;
    d_Sift[i + 4 * maxPts] = __fdividef(tra * tra, det);
}