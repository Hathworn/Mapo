#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int pitch, int height)
{
    // Data cache
    __shared__ float data1[3*(MINMAX_W + 2)];
    __shared__ float data2[3*(MINMAX_W + 2)];
    __shared__ float data3[3*(MINMAX_W + 2)];
    __shared__ float ymin1[(MINMAX_W + 2)];
    __shared__ float ymin2[(MINMAX_W + 2)];
    __shared__ float ymin3[(MINMAX_W + 2)];
    __shared__ float ymax1[(MINMAX_W + 2)];
    __shared__ float ymax2[(MINMAX_W + 2)];
    __shared__ float ymax3[(MINMAX_W + 2)];

    // Current tile and apron limits, relative to row start
    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_W;
    const int xEndClamped = min(xStart + MINMAX_W - 1, width - 1);
    int memWid = MINMAX_W + 2;

    int memPos0 = (tx - WARP_SIZE + 1);
    int memPos1 = (tx - WARP_SIZE + 1);
    int yq = 0;
    unsigned int output = 0;

    for (int y = 0; y < 32 + 2; y++) {
        output >>= 1;
        int memPos = yq * memWid + memPos0;
        int yp = 32 * blockIdx.y + y - 1;
        yp = max(yp, 0);
        yp = min(yp, height - 1);
        int readStart = yp * pitch;

        // Set the entire data cache contents
        if (tx >= (WARP_SIZE - 1)) {
            if (tx + xStart - WARP_SIZE < 0 || tx + xStart - WARP_SIZE >= width) {
                data1[memPos] = 0;
                data2[memPos] = 0;
                data3[memPos] = 0;
            } else {
                data1[memPos] = d_Data1[readStart + tx + xStart - WARP_SIZE];
                data2[memPos] = d_Data2[readStart + tx + xStart - WARP_SIZE];
                data3[memPos] = d_Data3[readStart + tx + xStart - WARP_SIZE];
            }
        }
        __syncthreads();

        int memPos2 = yq * memWid + tx;

        // Compute 3D minimum and maximum values
        if (y > 1 && tx < memWid) {
            float min1 = fminf(fminf(data1[memPos0], data1[memPos1]), data1[memPos2]);
            float min2 = fminf(fminf(data2[memPos0], data2[memPos1]), data2[memPos2]);
            float min3 = fminf(fminf(data3[memPos0], data3[memPos1]), data3[memPos2]);
            float max1 = fmaxf(fmaxf(data1[memPos0], data1[memPos1]), data1[memPos2]);
            float max2 = fmaxf(fmaxf(data2[memPos0], data2[memPos1]), data2[memPos2]);
            float max3 = fmaxf(fmaxf(data3[memPos0], data3[memPos1]), data3[memPos2]);
            ymin1[tx] = min1;
            ymin2[tx] = fminf(fminf(min1, min2), min3);
            ymin3[tx] = min3;
            ymax1[tx] = max1;
            ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
            ymax3[tx] = max3;
        }
        __syncthreads();

        // Evaluate threshold conditions
        if (y > 1 && tx < MINMAX_W && (tx + xStart) <= xEndClamped) {
            float minv = fminf(fminf(ymin2[tx], ymin2[tx+2]), ymin1[tx+1]);
            minv = fminf(minv, fminf(ymin3[tx+1], data2[memPos0+1]));
            minv = fminf(minv, d_Threshold[1]);
            float maxv = fmaxf(fmaxf(ymax2[tx], ymax2[tx+2]), ymax1[tx+1]);
            maxv = fmaxf(maxv, fmaxf(ymax3[tx+1], data2[memPos0+1]));
            maxv = fmaxf(maxv, d_Threshold[0]);
            if (data2[memPos1+1] < minv || data2[memPos1+1] > maxv)
                output |= 0x80000000;
        }
        __syncthreads();

        memPos0 = memPos1;
        memPos1 = memPos2;
        yq = (yq < 2 ? yq + 1 : 0);
    }
    
    // Write results
    if (tx < MINMAX_W && (tx + xStart) < width) {
        int writeStart = blockIdx.y * pitch + (tx + xStart);
        d_Result[writeStart] = output;
    }
}