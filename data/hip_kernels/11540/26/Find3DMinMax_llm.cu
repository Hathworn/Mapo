#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int height) {
    // Use dynamic shared memory declaration
    extern __shared__ float sharedData[];

    // Data cache allocation in shared memory
    float* data1 = sharedData;
    float* data2 = &sharedData[3 * (MINMAX_SIZE + 2)];
    float* data3 = &sharedData[6 * (MINMAX_SIZE + 2)];
    float* ymin1 = &sharedData[9 * (MINMAX_SIZE + 2)];
    float* ymin2 = &sharedData[10 * (MINMAX_SIZE + 2)];
    float* ymin3 = &sharedData[11 * (MINMAX_SIZE + 2)];
    float* ymax1 = &sharedData[12 * (MINMAX_SIZE + 2)];
    float* ymax2 = &sharedData[13 * (MINMAX_SIZE + 2)];
    float* ymax3 = &sharedData[14 * (MINMAX_SIZE + 2)];

    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_SIZE;
    const int xEnd = xStart + MINMAX_SIZE - 1;
    const int xReadPos = xStart + tx - WARP_SIZE;
    const int xWritePos = xStart + tx;
    const int xEndClamped = min(xEnd, width - 1);
    int memWid = MINMAX_SIZE + 2;
    int memPos0 = (tx - WARP_SIZE + 1);
    int memPos1 = (tx - WARP_SIZE + 1);
    int yq = 0;
    unsigned int output = 0;

    for (int y = 0; y < 34; ++y) {
        output >>= 1;
        
        // Use offsets instead of individual loop variable to reduce algebra operations
        int yp = max(32 * blockIdx.y + y - 1, 0);
        yp = min(yp, height - 1);
        int readStart = __mul24(yp, width);

        int memPos = yq * memWid + memPos0;

        // Unified loading condition to reduce instruction divergence 
        if (tx >= (WARP_SIZE - 1)) {
            float value1 = (xReadPos >= 0 && xReadPos < width) ? d_Data1[readStart + xReadPos] : 0;
            float value2 = (xReadPos >= 0 && xReadPos < width) ? d_Data2[readStart + xReadPos] : 0;
            float value3 = (xReadPos >= 0 && xReadPos < width) ? d_Data3[readStart + xReadPos] : 0;
            data1[memPos] = value1;
            data2[memPos] = value2;
            data3[memPos] = value3;
        }
        __syncthreads();

        int memPos2 = yq * memWid + tx;

        if (y > 1 && tx < memWid) {  // Combine condition to reduce __syncthreads calls
            float min1 = fminf(fminf(data1[memPos0], data1[memPos1]), data1[memPos2]);
            float min2 = fminf(fminf(data2[memPos0], data2[memPos1]), data2[memPos2]);
            float min3 = fminf(fminf(data3[memPos0], data3[memPos1]), data3[memPos2]);
            float max1 = fmaxf(fmaxf(data1[memPos0], data1[memPos1]), data1[memPos2]);
            float max2 = fmaxf(fmaxf(data2[memPos0], data2[memPos1]), data2[memPos2]);
            float max3 = fmaxf(fmaxf(data3[memPos0], data3[memPos1]), data3[memPos2]);

            ymin1[tx] = min1;
            ymin2[tx] = fminf(fminf(min1, min2), min3);
            ymin3[tx] = min3;
            ymax1[tx] = max1;
            ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
            ymax3[tx] = max3;
        }
        __syncthreads();

        if (y > 1 && tx < MINMAX_SIZE && xWritePos <= xEndClamped) {  // Combine conditions to reduce checks
            float minv = fminf(fminf(fminf(fminf(fminf(ymin2[tx], ymin2[tx + 2]), ymin1[tx + 1]),
                                          ymin3[tx + 1]), data2[memPos0 + 1]), data2[memPos2 + 1]);
            minv = fminf(minv, d_ConstantA[1]);

            float maxv = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(ymax2[tx], ymax2[tx + 2]), ymax1[tx + 1]),
                                          ymax3[tx + 1]), data2[memPos0 + 1]), data2[memPos2 + 1]);
            maxv = fmaxf(maxv, d_ConstantA[0]);

            if (data2[memPos1 + 1] < minv || data2[memPos1 + 1] > maxv) {
                output |= 0x80000000;
            }
        }
        __syncthreads();

        memPos0 = memPos1;
        memPos1 = memPos2;
        yq = (yq < 2 ? yq + 1 : 0);
    }

    if (tx < MINMAX_SIZE && xWritePos < width) {
        int writeStart = __mul24(blockIdx.y, width) + xWritePos;
        d_Result[writeStart] = output;
    }
}