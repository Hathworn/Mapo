#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOrientations(float *g_Data, float *d_Sift, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Initialize histogram bins to zero
    for (int i = 0; i < 13; i++)
        hist[i * 32 + tx] = 0.0f;
    __syncthreads();

    // Precompute Gaussian weights
    if (tx < 15) {
        float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));
    }

    int xp = (int)(d_Sift[bx + 0 * maxPts] - 6.5f);
    int yp = (int)(d_Sift[bx + 1 * maxPts] - 6.5f);
    int px = xp & 15;
    int x = tx - px;

    // Load data into shared memory with boundary checks
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = max(0, min(w - 1, xp + x)); // Clamp xi
        int yi = max(0, min(h - 1, yp + y)); // Clamp yi
        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    // Compute gradients and organize into histogram
    if (x >= 1 && x < 14) {
        for (int y = 1; y < 14; y++) {
            int memPos = 16 * y + x;
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = 16.0f * atan2f(dy, dx) / 3.1416f + 16.5f;
            bin = bin < 32 ? bin : 0; // Adjust for bin overflow
            float grad = sqrtf(dx * dx + dy * dy);
            atomicAdd(&hist[32 * (x - 1) + bin], grad * gauss[x] * gauss[y]); // Use atomic addition
        }
    }
    __syncthreads();

    // Reduce histogram bins
    for (int offset = 8; offset > 0; offset >>= 1) {
        for (int y = 0; y < 13 - offset; y += offset)
            hist[y * 32 + tx] += hist[(y + offset) * 32 + tx];
        __syncthreads();
    }
    
    // Smoothing histogram and finding peaks
    if (tx < 32) {
        hist[tx] = hist[32 * 0 + tx] * 6 + (hist[32 * 1 + (tx + 1) % 32] + hist[32 * 1 + (tx + 31) % 32]) * 4 + (hist[32 * 1 + (tx + 2) % 32] + hist[32 * 1 + (tx + 30) % 32]);
        __syncthreads();

        float v = hist[tx];
        float v_right = hist[(tx + 1) % 32];
        float v_left = hist[(tx + 31) % 32];

        if (v > v_right && v >= v_left) {
            float peak = tx + 0.5f * (v_right - v_left) / (2.0f * v - v_right - v_left);
            if (tx == 0) {
                d_Sift[bx + 5 * maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
                // Determine second peak if valid
                float maxval = v * 0.8f;
                for (int i = 1; i < 32; i++) {
                    if (hist[i] > maxval) {
                        float v1 = hist[(i + 1) % 32];
                        float v2 = hist[(i + 31) % 32];
                        peak = i + 0.5f * (v1 - v2) / (2.0f * hist[i] - v1 - v2);
                        d_Sift[bx + 6 * maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
                        break;
                    }
                }
            }
        }
    }
}