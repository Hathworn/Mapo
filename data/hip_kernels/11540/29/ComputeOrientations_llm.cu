#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Define block size for performance optimization
#define BLOCK_SIZE 32 

__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[BLOCK_SIZE*14];
    __shared__ float gauss[16];

    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Initialize histogram to zero
    for (int i = 0; i < 13; i++)
        hist[i * BLOCK_SIZE + tx] = 0.0f;
    __syncthreads();

    // Precompute Gaussian values
    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    if (tx < 15)
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));

    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;
    int px = xp & 15;
    int x = tx - px;

    // Load data and apply boundary conditions
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = xp + x;
        int yi = yp + y;
        if (xi < 0) xi = 0;
        if (xi >= w) xi = w - 1;
        if (yi < 0) yi = 0;
        if (yi >= h) yi = h - 1;
        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    // Compute gradients and update histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            float bin = 16.0f * atan2f(dy, dx) / 3.1416f + 16.5f;
            if (bin == 32)
                bin = 0;
            float grad = sqrtf(dx * dx + dy * dy);
            hist[32 * (x - 1) + static_cast<int>(bin)] += grad * gauss[x] * gauss[y];
        }
    }
    __syncthreads();

    // Reduce histogram values
    for (int offset = 8; offset >= 1; offset >>= 1) {
        for (int y = 0; y < BLOCK_SIZE / (2 * offset); y++)
            hist[y * BLOCK_SIZE + tx] += hist[(y + offset) * BLOCK_SIZE + tx];
        __syncthreads();
    }

    // Apply smoothing
    if (tx < BLOCK_SIZE) {
        int prev = (tx + BLOCK_SIZE - 1) % BLOCK_SIZE;
        int next = (tx + 1) % BLOCK_SIZE;
        hist[BLOCK_SIZE + tx] = 6 * hist[tx] + 4 * (hist[next] + hist[prev]) + (hist[(next + 1) % BLOCK_SIZE] + hist[(prev + BLOCK_SIZE - 1) % BLOCK_SIZE]);
    }
    __syncthreads();

    // Find peak values for orientation
    if (tx == 0) {
        float maxval1 = 0.0f, maxval2 = 0.0f;
        int i1 = -1, i2 = -1;
        for (int i = 0; i < BLOCK_SIZE; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }

        // Parabolic interpolation for accurate peak position
        float val1 = hist[BLOCK_SIZE + ((i1 + 1) & (BLOCK_SIZE - 1))];
        float val2 = hist[BLOCK_SIZE + ((i1 + BLOCK_SIZE - 1) & (BLOCK_SIZE - 1))];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Orient[bx] = 11.25f * (peak < 0.0f ? peak + BLOCK_SIZE : peak);

        if (maxval2 < 0.8f * maxval1)
            i2 = -1;
        if (i2 >= 0) {
            val1 = hist[BLOCK_SIZE + ((i2 + 1) & (BLOCK_SIZE - 1))];
            val2 = hist[BLOCK_SIZE + ((i2 + BLOCK_SIZE - 1) & (BLOCK_SIZE - 1))];
            peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Orient[bx + maxPts] = 11.25f * (peak < 0.0f ? peak + BLOCK_SIZE : peak);
        } else
            d_Orient[bx + maxPts] = i2;
    }
}