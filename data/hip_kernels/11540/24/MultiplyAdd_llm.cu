#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify indexing
    const int y = blockIdx.y * blockDim.y + threadIdx.y;  // Simplify indexing
    int p = y * width + x;  // Calculate linear index

    if (x < width && y < height) {
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
}