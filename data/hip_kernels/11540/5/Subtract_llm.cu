#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    // Calculate the global thread index for x and y dimensions
    const int x = blockIdx.x * SUBTRACT_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACT_H + threadIdx.y;
    
    // Calculate the linear index
    int p = y * pitch + x;
    
    // Check bounds and perform subtraction if within bounds
    if (x < width && y < height)
    {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // Removed unnecessary __syncthreads() due to no shared memory usage
}