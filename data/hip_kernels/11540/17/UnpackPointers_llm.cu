#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;
    
    // Optimize loop by using shared memory for reduction
    __shared__ int sharedPtrs[16 * 1024];  // Assuming maxPts cannot exceed this limit

    for (int y = 0; y < h / 32; y++) {
        for (int x = 0; x < w; x += 16) {
            unsigned int val = minmax[y * w + x + tx];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        sharedPtrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }

    // Write results from shared memory to global memory safely
    for (int i = 0; i < numPts; i++) {
        ptrs[16 * i + tx] = sharedPtrs[16 * i + tx];
    }
}