#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Use shared memory to reduce global memory accesses (not strongly beneficial here)
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Unrolling loop to enhance performance
    for (unsigned int i = idx; i < numEls; i += numThreads * 4) {
        if (i < numEls) dest[i] = a[i] + b[i];
        if (i + numThreads < numEls) dest[i + numThreads] = a[i + numThreads] + b[i + numThreads];
        if (i + numThreads * 2 < numEls) dest[i + numThreads * 2] = a[i + numThreads * 2] + b[i + numThreads * 2];
        if (i + numThreads * 3 < numEls) dest[i + numThreads * 3] = a[i + numThreads * 3] + b[i + numThreads * 3];
    }
}