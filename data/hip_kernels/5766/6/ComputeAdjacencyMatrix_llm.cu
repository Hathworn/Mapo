#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeAdjacencyMatrix(float* dOut, int* nn, int n, int k)
{
    // Calculate global thread ID
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds
    if (col < n)
    {
        int* nnCol = &nn[col * n];
        int baseIndex = col * n;
        for (int i = 0; i < k; ++i)
        {
            int neighborIdx = nnCol[i];
            int idx1 = baseIndex + neighborIdx;
            int idx2 = neighborIdx * n + col;
            // Set adjacency matrix entries
            dOut[idx1] = 1.0f;
            dOut[idx2] = 1.0f;
        }
    }
}