#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SortDistances(float* dist, int* idMat, int n, int k)
{
    // Get the index of the column that the current thread is responsible for
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // IF col is out of bounds, then do nothing
    if(col < n)
    {
        int* id = &idMat[col * n];
        for(int i = 0; i < n; ++i)
            id[i] = i;

        float* distCol = &dist[col * n];
        // Only care about the first k elements being sorted
        for (int i = 0; i < k; ++i)
        {
            int minIndex = i;
            float minValue = distCol[i];

            // Optimize by reducing redundant memory access
            for (int j = i + 1; j < n; ++j)
            {
                if(distCol[j] < minValue)
                {
                    minIndex = j;
                    minValue = distCol[j];
                }
            }
            if(minIndex != i)
            {
                // Swap distCol
                distCol[minIndex] = distCol[i];
                distCol[i] = minValue;

                // Swap ids
                int tmpId = id[minIndex];
                id[minIndex] = id[i];
                id[i] = tmpId;
            }
        }
    }
}