#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InsertionSortDistances(float* dist, int* idMat, int n, int k)
{
    // Get the index of the column that the current thread is responsible for
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // If col is out of bounds, then do nothing
    if (col < n)
    {
        int* id = &idMat[col * n];
        id[0] = 0;

        float* distCol = &dist[col * n];

        // Optimized sorting of column 'col'
        for (int i = 1; i < n; ++i)
        {
            float x = distCol[i];
            int currIndex = i;
            int j = i - 1;
            while (j >= 0 && distCol[j] > x)
            {
                distCol[j + 1] = distCol[j];
                id[j + 1] = id[j];
                --j;
            }
            distCol[j + 1] = x;
            id[j + 1] = currIndex;
        }
    }
}