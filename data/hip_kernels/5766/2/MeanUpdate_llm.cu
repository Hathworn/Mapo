#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Start Header
***************************************************************** /
/*!
\file knn-kernel.cu
\author Koh Wen Lin
\brief
Contains the implementation for kmeans clustering on the gpu.
*/
/* End Header
*******************************************************************/
#define KMEAN_BLOCK_SIZE 32
#define KMEAN_BLOCK_SIZE_1D KMEAN_BLOCK_SIZE * KMEAN_BLOCK_SIZE

__global__ void MeanUpdate(float* dMeanIn, unsigned k, unsigned d, int* count)
{
    // Load count into a local variable to avoid repeated global memory access
    int idx = threadIdx.x;
    int localCount = max(1, count[idx]);
    float ooc = 1.0f / localCount;

    // Use loop unrolling for faster memory access and processing
    #pragma unroll
    for(int i = 0; i < d; ++i)
    {
        dMeanIn[idx * d + i] *= ooc;
    }
}