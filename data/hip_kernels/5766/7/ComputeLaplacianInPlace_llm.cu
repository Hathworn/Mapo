#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeLaplacianInPlace(float* d, int n)
{
    // Calculate global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within the matrix bounds
    if(x < n)
    {
        auto dCol = &d[x * n];
        float sum = 0;

        // Loop unrolling and calculate column sum
        for(int i = 0; i < n; i += 2)
        {
            if(i != x) 
            {
                sum += dCol[i];
                dCol[i] = -dCol[i];
            }
            if(i + 1 != x && i + 1 < n) 
            {
                sum += dCol[i + 1];
                dCol[i + 1] = -dCol[i + 1];
            }
        }
        
        // Separate updates outside the loop to prevent race conditions
        dCol[x] += sum;
    }
}