#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeSquareDistance(float* dOut, float* dIn, int n, int d)
{
    // Use shared memory to store blocks of data
    __shared__ float blockA[KNN_BLOCK_SIZE][KNN_BLOCK_SIZE];
    __shared__ float blockB[KNN_BLOCK_SIZE][KNN_BLOCK_SIZE];

    // Determine starting indices for blocks A and B
    int aStart = blockIdx.x * blockDim.x;
    int bStart = blockIdx.y * blockDim.y;

    int ax = aStart + threadIdx.x;
    int bx = bStart + threadIdx.y;

    float sqDist = 0.0f;

    // Calculate number of vertical blocks needed
    int numBlocksVertical = (d + KNN_BLOCK_SIZE - 1) / KNN_BLOCK_SIZE;

    for(int i = 0; i < numBlocksVertical; ++i)
    {
        int startY = i * KNN_BLOCK_SIZE;
        int currY  = startY + threadIdx.y;

        // Load data into shared memory
        if(startY + threadIdx.y < d)
        {
            if(ax < n)
                blockA[threadIdx.y][threadIdx.x] = dIn[ax * d + currY];
            if(bx < n)
                blockB[threadIdx.y][threadIdx.x] = dIn[(bStart + threadIdx.x) * d + currY];
        }
        __syncthreads();

        // Compute partial square distances
        if(ax < n && bx < n)
        {
            #pragma unroll  // Unroll loop for performance
            for(int j = 0; j < KNN_BLOCK_SIZE; ++j)
            {
                float diff = blockA[j][threadIdx.x] - blockB[j][threadIdx.y];
                sqDist += diff * diff;
            }
        }
        __syncthreads();
    }

    // Output the computed square distance
    if(ax < n && bx < n)
    {
        dOut[ax * n + bx] = ax == bx ? INFINITY : sqDist;
    }
}