#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Start Header
***************************************************************** /
/*!
\file knn-kernel.cu
\author Koh Wen Lin
\brief
Contains the implementation for kmeans clustering on the gpu.
*/
/* End Header
*******************************************************************/
#define KMEAN_BLOCK_SIZE 32
#define KMEAN_BLOCK_SIZE_1D KMEAN_BLOCK_SIZE * KMEAN_BLOCK_SIZE

__global__ void Mean(float* dIn, unsigned n, unsigned d, int* dGroupIn, float* dMeanIn, unsigned k, int* count) {
    // Shared memory allocation
    extern __shared__ float sDataSumGroupCount[];

    float* sData = sDataSumGroupCount;
    float* sSum = sData + KMEAN_BLOCK_SIZE_1D * d;
    int* sGroup = (int*)&sDataSumGroupCount[(k + KMEAN_BLOCK_SIZE_1D) * d];
    int* sCount = sGroup + KMEAN_BLOCK_SIZE_1D;

    const int tx = threadIdx.x;
    int tid = blockIdx.x * blockDim.x + tx;

    if (tid >= n) return;

    // Initialize shared memory
    if (tx < k) {
        for (int i = 0; i < d; ++i) 
            sSum[tx * d + i] = dMeanIn[tx * d + i];
        sCount[tx] = 0.0f;
    }

    __syncthreads(); // Make sure the initialization is complete before proceeding

    // Load data and group index
    memcpy(&sData[tx * d], &dIn[tid * d], d * sizeof(float));
    sGroup[tx] = dGroupIn[tid];

    __syncthreads(); // Ensure all data relevant to block is loaded

    int clusterId = sGroup[tx];

    // Atomic operations for safe concurrent writing
    for (int i = 0; i < d; ++i)
        atomicAdd(&sSum[clusterId * d + i], sData[tx * d + i]);
    atomicAdd(&sCount[clusterId], 1);

    __syncthreads(); // Ensure all atomic additions are completed before proceeding

    // Accumulate results in global memory
    if (tx == 0) {
        for (int i = 0; i < k * d; ++i)
            atomicAdd(&dMeanIn[i], sSum[i]);

        for (int i = 0; i < k; ++i)
            atomicAdd(&count[i], sCount[i]);
    }
}