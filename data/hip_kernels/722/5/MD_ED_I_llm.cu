#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MD_ED_I(float *S, float *T, int trainSize, int window_size, int dimensions, float *data_out, int task, int gm) {

    int idx, offset_x;
    float sumErr = 0;
    long long int i, j;

    if(gm == 0){

        extern __shared__ float sh_mem[];

        float *T2 = (float *)sh_mem;
        float *DTW_single_dim = (float *)&sh_mem[dimensions * window_size];

        if (task == 0) {
            idx = threadIdx.x * dimensions + threadIdx.y;
            offset_x = ((blockDim.x * blockDim.y * window_size) * blockIdx.x) + idx * window_size;

            if (((blockDim.x * blockDim.y * blockIdx.x) + idx) >= trainSize * dimensions)
                return;

        } else { // SUBSEQ_SEARCH
            idx = threadIdx.x * dimensions + threadIdx.y;
            offset_x = (blockDim.x * blockIdx.x) + ((threadIdx.y * trainSize) + threadIdx.x);

            if ((idx + window_size) > trainSize)
                return;
        }

        // Preload data into shared memory
        if (idx < dimensions * window_size) {
            T2[idx] = T[idx];
        }
        __syncthreads();

        // Reduce the number of stores to shared memory
        for (j = 0; j < window_size; j++) {
            float diff = S[offset_x + j] - T2[window_size * threadIdx.y + j];
            sumErr += diff * diff;
        }
        DTW_single_dim[idx] = sqrt(sumErr);

        __syncthreads();

        // Reduce threads writing to global memory; use single thread to sum up results
        if (idx == 0) {
            for (i = 0; i < blockDim.x; i++) {
                float blockSum = 0.0;
                for (j = 0; j < blockDim.y; j++) {
                    blockSum += DTW_single_dim[i * dimensions + j];
                }
                data_out[(blockIdx.x * blockDim.x) + i] = blockSum;
            }
        }

    } else {

        extern __shared__ float DTW_single_dim[];

        if (task == 0) {
            idx = threadIdx.x * dimensions + threadIdx.y;
            offset_x = ((blockDim.x * blockDim.y * window_size) * blockIdx.x) + idx * window_size;

            if (((blockDim.x * blockDim.y * blockIdx.x) + idx) >= trainSize * dimensions)
                return;

        } else { // SUBSEQ_SEARCH
            idx = threadIdx.x * dimensions + threadIdx.y;
            offset_x = (blockDim.x * blockIdx.x) + ((threadIdx.y * trainSize) + threadIdx.x);

            if ((idx + window_size) > trainSize)
                return;
        }

        // Direct use of global memory for T
        for (j = 0; j < window_size; j++) {
            float diff = S[offset_x + j] - T[window_size * threadIdx.y + j];
            sumErr += diff * diff;
        }
        DTW_single_dim[idx] = sqrt(sumErr);

        __syncthreads();

        if (idx == 0) {
            for (i = 0; i < blockDim.x; i++) {
                float blockSum = 0.0;
                for (j = 0; j < blockDim.y; j++) {
                    blockSum += DTW_single_dim[i * dimensions + j];
                }
                data_out[(blockIdx.x * blockDim.x) + i] = blockSum;
            }
        }

    }

}