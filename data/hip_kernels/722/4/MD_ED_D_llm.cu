#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MD_ED_D(float *S, float *T, int trainSize, int window_size, int dimensions, float *data_out, int task, int gm) {

    long long int i, j, p;
    float sumErr = 0, dd = 0;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce redundant global memory accesses
    extern __shared__ float T2[];

    if (gm == 0) {
        int t, offset;
        if (task == 0) {
            offset = window_size;
            int wind = dimensions * window_size;
            t = idx * wind;
            if ((idx * wind) + wind > trainSize * wind) return;

            // Load data into shared memory once per block to minimize memory transactions
            if (threadIdx.x == 0) {
                for (i = 0; i < dimensions; i++)
                    for (j = 0; j < window_size; j++)
                        T2[window_size * i + j] = T[window_size * i + j];
            }

            __syncthreads();
        } else {
            offset = trainSize;
            t = idx;
            if ((idx + window_size) > trainSize) return;

            // Load data into shared memory once per block to minimize memory transactions
            if (threadIdx.x == 0) {
                for (i = 0; i < dimensions; i++)
                    for (j = 0; j < window_size; j++)
                        T2[window_size * i + j] = T[window_size * i + j];
            }
            __syncthreads();
        }

        // Optimize calculation using shared memory
        for (j = 0; j < window_size; j++) {
            dd = 0;
            for (p = 0; p < dimensions; p++) {
                float diff = S[(t + p * offset) + j] - T2[(p * window_size) + j];
                dd += diff * diff;
            }
            sumErr += dd;
        }
        data_out[idx] = sqrt(sumErr);
    } else {
        int t, offset;
        if (task == 0) {
            offset = window_size;
            int wind = dimensions * window_size;
            t = idx * wind;
            if ((idx * wind) + wind > trainSize * wind) return;
        } else {
            offset = trainSize;
            t = idx;
            if ((idx + window_size) > trainSize) return;
        }

        for (j = 0; j < window_size; j++) {
            dd = 0;
            for (p = 0; p < dimensions; p++) {
                float diff = S[(t + p * offset) + j] - T[(p * window_size) + j];
                dd += diff * diff;
            }
            sumErr += dd;
        }
        data_out[idx] = sqrt(sumErr);
    }
}