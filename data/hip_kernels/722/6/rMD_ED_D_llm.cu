#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rMD_ED_D(float *S, float *T, int window_size, int dimensions, float *data_out, int trainSize, int gm) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit if index is out of bounds
    if (idx >= (trainSize * window_size))
        return;

    extern __shared__ float T2[];

    int s = dimensions * 2 * window_size * (idx / window_size);
    int t = s + idx % window_size;
    float sumErr = 0.0f;

    if (gm == 0) {

        // Load T to shared memory once per block
        if (threadIdx.x < dimensions * window_size) {
            T2[threadIdx.x] = T[threadIdx.x];
        }
        __syncthreads();

        // Calculate error for shared memory case
        for (int j = 0; j < window_size; j++) {
            float dd = 0.0f;
            for (int p = 0; p < dimensions; p++) {
                float diff = S[(t + p * 2 * window_size) + j] - T2[(p * window_size) + j];
                dd += diff * diff;
            }
            sumErr += dd;
        }
        
    } else {

        // Calculate error for non-shared memory case
        for (int j = 0; j < window_size; j++) {
            float dd = 0.0f;
            for (int p = 0; p < dimensions; p++) {
                float diff = S[(t + p * 2 * window_size) + j] - T[(p * window_size) + j];
                dd += diff * diff;
            }
            sumErr += dd;
        }
        
    }

    // Store the result
    data_out[idx] = sqrt(sumErr);
}