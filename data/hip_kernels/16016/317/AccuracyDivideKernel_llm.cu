#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Divide accuracy by N using a single thread to prevent data races
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {  // Ensure only one thread performs the division
        *accuracy /= N;
    }
}