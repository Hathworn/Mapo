#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiply(double *a, double *b, double *c, int cr, int cc, int ac, int bc){

    // Calculate global row (y) and column (x) index for the thread
    long x = blockIdx.x * blockDim.x + threadIdx.x;
    long y = blockIdx.y * blockDim.y + threadIdx.y;

    // Initialize sum
    double sum = 0;

    // Check if within matrix bounds
    if(x < cc && y < cr){
        
        // Use shared memory for common elements
        extern __shared__ double shared[];
        double *shared_a = shared;
        double *shared_b = shared + blockDim.y * ac;

        // Loop over all elements in the current row/column pair
        for(int k = 0; k < ac; k++){

            // Load data into shared memory
            shared_a[threadIdx.y * ac + k] = a[y * ac + k];
            shared_b[k * bc + threadIdx.x] = b[k * bc + x];
            __syncthreads();

            // Perform partial dot product
            sum += shared_a[threadIdx.y * ac + k] * shared_b[k * bc + threadIdx.x];
            __syncthreads();
        }
        
        // Write result back to global memory
        c[y * cc + x] = sum;
    }
}