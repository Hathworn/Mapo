#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixSubScalar(double *a, double b, double *c, int cr, int cc) {
    // Calculate the linear index once
    int idx = blockIdx.y * blockDim.y * cc + blockIdx.x * blockDim.x + threadIdx.y * cc + threadIdx.x;

    // Ensure work is within bounds
    if (threadIdx.y + blockIdx.y * blockDim.y < cr && threadIdx.x + blockIdx.x * blockDim.x < cc) {
        c[idx] = a[idx] - b;
    }
}