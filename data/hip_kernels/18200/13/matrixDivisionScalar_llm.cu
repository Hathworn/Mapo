#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixDivisionScalar(double *a, double b, double *c, int cr, int cc) {
    // Calculate the global thread index
    int idx = blockIdx.y * blockDim.y * cc + blockIdx.x * blockDim.x + threadIdx.y * cc + threadIdx.x; 

    // Use a single if condition to check within bounds
    if (idx < cr * cc) {
        c[idx] = a[idx] / b;
    }
}