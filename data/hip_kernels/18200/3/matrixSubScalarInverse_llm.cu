#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixSubScalarInverse(double *a, double b, double *c, int cr, int cc){
    int x = blockIdx.x * blockDim.x + threadIdx.x; // col
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    // Unroll loops to improve memory access pattern
#pragma unroll
    for(int i = 0; i < 1; ++i){
        if(x < cc && y < cr){
            c[y * cc + x] = b - a[y * cc + x];
        }
    }
}