#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixClip(double *a, double min, double max, double *c, int cr, int cc) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // col
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    if (x < cc && y < cr) {
        double val = a[y * cc + x]; // Load value once
        c[y * cc + x] = (val > max) ? max : ((val < min) ? min : val); // Simplified conditional
    }
}