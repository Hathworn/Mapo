#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixHadamard(double *a, double *b, double *c, int cr, int cc) {
    // Calculate the unique thread index directly
    int idx = blockIdx.y * blockDim.y * cc + blockIdx.x * blockDim.x + threadIdx.y * cc + threadIdx.x;

    // Ensure index is within bounds of the matrices
    if (idx < cr * cc) {
        c[idx] = a[idx] * b[idx];
    }
}