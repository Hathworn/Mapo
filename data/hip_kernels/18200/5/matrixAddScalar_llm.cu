#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixAddScalar(double *a, double b, double *c, int cr, int cc) {
    // Calculate the global index for current thread
    int index = blockIdx.y * blockDim.y * cc + blockIdx.x * blockDim.x + threadIdx.y * cc + threadIdx.x;
    
    // Perform addition only if index is within bounds
    if (index < cr * cc) {
        c[index] = a[index] + b; // Direct access using global index
    }
}