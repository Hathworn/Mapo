#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixSub(double *a, double *b, double *c, int cr, int cc) {
    // Calculate thread's unique row and column index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = y * cc + x; // Pre-compute index for better performance

    if (x < cc && y < cr) {
        c[idx] = a[idx] - b[idx];
    }
}