#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixExp(double *a, double *c, int cr, int cc) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // col
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    // Use shared memory to store data temporarily for faster access
    extern __shared__ double shared_a[];

    if (x < cc && y < cr) {
        // Load data into shared memory
        int index = y * cc + x;
        shared_a[threadIdx.y * blockDim.x + threadIdx.x] = a[index];
        __syncthreads();

        // Perform the computation using shared memory
        c[index] = exp(shared_a[threadIdx.y * blockDim.x + threadIdx.x]);
    }
}