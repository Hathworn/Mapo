#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMultiplyScalar(double *a, double b, double *c, int cr, int cc) {
    // Calculate the global thread index in the matrix
    int idx = blockIdx.y * blockDim.y * cc + blockIdx.x * blockDim.x + threadIdx.y * cc + threadIdx.x;
    
    // Ensure the thread operates only within bounds
    if (idx < cr * cc) {
        c[idx] = a[idx] * b; // Perform scalar multiplication
    }
}