#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixAdd(double *a, double *b, double *c, int cr, int cc) {
    // Calculate a single index to reduce the number of computations
    long idx = (blockIdx.y * blockDim.y + threadIdx.y) * cc + (blockIdx.x * blockDim.x + threadIdx.x);
    
    // Use an inline check to ensure both x and y are within bounds
    if (idx < cr * cc) {
        c[idx] = a[idx] + b[idx];
    }
}