#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixSigmoid(double *a, double *c, int cr, int cc) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // col
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    if (x < cc && y < cr) {
        double a_val = a[y * cc + x]; // Reduce redundant memory access
        c[y * cc + x] = 1.0 / (1.0 + exp(-a_val));
    }
}