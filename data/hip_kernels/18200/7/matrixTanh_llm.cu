#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixTanh(double *a, double *c, int cr, int cc) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // col
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    // Use a single index calculation to help compiler optimization
    int index = y * cc + x;
    
    if (x < cc && y < cr) {
        c[index] = tanh(a[index]);
    }
}