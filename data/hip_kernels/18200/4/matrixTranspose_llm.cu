#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixTranspose(double *a, double *c, int cr, int cc) {

    int x = blockIdx.x * blockDim.x + threadIdx.x; // col
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    if(x < cc && y < cr) {
        // Use shared memory to optimize memory access
        __shared__ double tile[32][32 + 1]; // Assuming block size 32x32; add padding to avoid bank conflicts

        tile[threadIdx.y][threadIdx.x] = a[y * cc + x];

        __syncthreads();

        c[x * cr + y] = tile[threadIdx.x][threadIdx.y]; // Transposed write with coalesced access
    }
}