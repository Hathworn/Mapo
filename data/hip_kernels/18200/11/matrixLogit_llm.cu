#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixLogit(double *a, double *c, int cr, int cc){

    int x = blockIdx.x * blockDim.x + threadIdx.x; // col
    int y = blockIdx.y * blockDim.y + threadIdx.y; // row

    // Ensure thread within matrix bounds
    if (x < cc && y < cr) {
        double val = a[y * cc + x]; // Load once
        c[y * cc + x] = log(val / (1 - val)); // Compute logit
    }
}