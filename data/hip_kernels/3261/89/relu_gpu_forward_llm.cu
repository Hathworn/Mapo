#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Compute 2D grid and block index to allow for more threads to utilize shared memory
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use a for loop to allow each thread to handle multiple elements
    for (int i = index; i < N; i += stride) {
        out[i] = in[i] > 0 ? in[i] : 0;
    }
}