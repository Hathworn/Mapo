#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Define constants for initialization values to avoid magic numbers
#define INITIAL_POS_X 2.0f
#define INITIAL_POS_Y 0.0f
#define INITIAL_POS_Z 0.0f
#define INITIAL_VEL 0.0f
#define INITIAL_AGE -1.0f
#define INITIAL_LIFE -1.0f
#define INITIAL_OPAFIX 0

__global__ void resetParticlesKernel(float3* pos, float3* vel, float* age, float* life, char* opafix, int nParts)
{
    unsigned int n = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check boundary condition
    if (n < nParts) {
        // Initialize positions, velocities, age, life and opafix
        pos[n] = make_float3(INITIAL_POS_X, INITIAL_POS_Y, INITIAL_POS_Z);
        vel[n] = make_float3(INITIAL_VEL, INITIAL_VEL, INITIAL_VEL);
        age[n] = INITIAL_AGE;
        life[n] = INITIAL_LIFE;
        opafix[n] = INITIAL_OPAFIX;
    }
}