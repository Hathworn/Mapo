#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure only the first thread performs the division
    if (idx == 0) {
        *accuracy /= N;
    }
}