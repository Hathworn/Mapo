#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeGradientCentralDiff(const float* __restrict__ similarities, float* __restrict__ gradient, const int* __restrict__ activeMask, int activeSlices, int slices, int p)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= activeSlices) return;
    
    int slice = activeMask[i];

    // Use a single memory read for similarities to minimize access
    float similarity_value = similarities[slice];
    float dx = similarity_value - similarities[slices + slice];

    // Use direct multiplication result storage
    gradient[p * slices + slice] = dx;
    
    // Optimize updating the gradient to avoid branching
    float dx2 = dx * dx;
    atomicAdd(&gradient[6 * slices + slice], dx2 * (p != 0) + dx2 * (p == 0)); 
}