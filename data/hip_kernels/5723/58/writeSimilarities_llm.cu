#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeSimilarities(const float* nvccResults, int* activelayers, int writestep, int writenum, float* similarities, int active_patches, int patches)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < active_patches)
    {
        float res = nvccResults[tid];
        int patch = activelayers[tid];

        // Use shared memory to optimize global memory access pattern
        __shared__ float sharedRes;
        sharedRes = res;
        __syncthreads();

        for (int i = 0; i < writenum; ++i)
        {
            // Minimize arithmetic operations within the loop
            similarities[(i * writestep * patches) + patch] = sharedRes;
        }
    }
}