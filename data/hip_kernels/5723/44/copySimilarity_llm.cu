#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copySimilarity(float* similarities, int active_slices, int slices, int* activeMask, int target, int source)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if within bounds
    if (i < active_slices) {
        int slice = activeMask[i];
        // Efficient memory access using slice index
        similarities[target * slices + slice] = similarities[source * slices + slice];
    }
}