#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void addNccValues(const float* __restrict__ prevData, float* __restrict__ result, int slices)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < slices)
    {
        float val1 = prevData[3 * tid + 1];
        float val2 = prevData[3 * tid + 2];
        
        // Precompute multiplications and square root for better performance
        float norm = val1 * val2;
        if (norm > 0)
        {
            float res = prevData[3 * tid] * rsqrtf(norm); // Using reciprocal square root for optimization
            atomicAdd(&result[tid], res); // Use atomic operation to avoid potential race conditions
        }
    }
}