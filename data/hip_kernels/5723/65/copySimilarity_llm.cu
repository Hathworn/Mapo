#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copySimilarity(float* similarities, int active_patches, int patches, int* activeMask, int target, int source)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < active_patches) {
        int patch = activeMask[i];
        // Use shared memory to reduce global memory traffic
        __shared__ int sharedPatch;
        sharedPatch = patch;
        __syncthreads();
        similarities[target * patches + sharedPatch] = similarities[source * patches + sharedPatch];
    }
}