#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeGradientCentralDiff(const float* __restrict__ similarities, float* __restrict__ gradient, const int* __restrict__ activeMask, const int activePatches, const int patches, const int p)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= activePatches)
        return;
    
    int patch = activeMask[i];
    float dx = similarities[patch] - similarities[patches + patch];

    // Use atomic operations to prevent race conditions and ensure correct computations
    gradient[p * patches + patch] = dx;
    if (p == 0)
        gradient[6 * patches + patch] = dx * dx;
    else
        atomicAdd(&gradient[6 * patches + patch], dx * dx);
}