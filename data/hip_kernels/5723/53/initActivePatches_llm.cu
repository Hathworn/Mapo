#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initActivePatches(int* buffer, int num)
{
    // Check if index is within bounds and perform operation
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < num)
        buffer[i] = i;
}