#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initActiveSlices(int* buffer, int num)
{
    // Calculate the global thread index
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // Use warp divergence to ensure efficient execution
    if (i < num)
    {
        // Optimize by reducing memory access overhead
        buffer[i] = threadIdx.x + blockIdx.x * blockDim.x;
    }
}