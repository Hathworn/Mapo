#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalizeGradient(float* gradient, int* activeMask, int activeSlices, int slices)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= activeSlices) return;

    int slice = activeMask[i];

    // Load and compute norm once per slice
    float norm = gradient[6 * slices + slice];
    if (norm > 0)
        norm = 1.0f / sqrtf(norm);

    #pragma unroll  // Optimize loop by unrolling
    for (int j = 0; j < 6; ++j)
        gradient[j * slices + slice] *= norm;
}