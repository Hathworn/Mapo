#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addNccValues(const float* prevData, float* result, int patches)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < patches)
    {
        // Move redundant calculations outside the if statement
        float value1 = prevData[3 * tid + 1];
        float value2 = prevData[3 * tid + 2];
        float norm = value1 * value2;
        float res = 0.0f;
        if (norm > 0.0f)
            res = prevData[3 * tid] / sqrtf(norm);
        // Use efficient memory access by using += only once
        result[tid] += res;
    }
}