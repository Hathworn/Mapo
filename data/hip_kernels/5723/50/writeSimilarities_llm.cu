#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeSimilarities(const float* nvccResults, int* activelayers, int writestep, int writenum, float* similarities, int active_slices, int slices)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread id is within bounds
    if (tid < active_slices)
    {
        float res = nvccResults[tid];
        int slice = activelayers[tid];
        
        // Use a pointer offset to access memory more efficiently
        float* simPtr = similarities + slice;
        for (int i = 0; i < writenum; ++i)
        {
            *(simPtr + slices * writestep * i) = res;
        }
    }
}