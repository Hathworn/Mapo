#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalizeGradient(float* gradient, int* activeMask, int activePatches, int patches)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= activePatches) return;

    int patch = activeMask[i];

    // Load norm value once to avoid multiple memory access
    float norm = gradient[6 * patches + patch];
    if (norm > 0)
        norm = 1.0f / sqrtf(norm);

    // Use loop unrolling for efficiency
    gradient[0 * patches + patch] *= norm;
    gradient[1 * patches + patch] *= norm;
    gradient[2 * patches + patch] *= norm;
    gradient[3 * patches + patch] *= norm;
    gradient[4 * patches + patch] *= norm;
    gradient[5 * patches + patch] *= norm;
}