#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Using thread 0 to perform division to avoid race conditions
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
    // Ensure all threads sync before exiting the kernel
    __syncthreads();
}