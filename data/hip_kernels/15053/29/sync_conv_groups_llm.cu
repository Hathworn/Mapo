#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups() {
    // Ensure all threads reach this point by using __syncthreads
    __syncthreads();
}