#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_image_array_GPU(unsigned long long int* image, int pixels_per_image)
{
    int my_pixel = threadIdx.x + blockIdx.x * blockDim.x;
    if (my_pixel < pixels_per_image)
    {
        // Use a loop to reduce repetitive code lines for initializing images
        for (int i = 0; i < 4; ++i)
        {
            image[my_pixel + i * pixels_per_image] = 0;
        }
    }
}