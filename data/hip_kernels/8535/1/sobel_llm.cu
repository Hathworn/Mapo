#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

typedef enum color {BLUE, GREEN, RED} Color;

__global__ void sobel(unsigned char *src, unsigned char *dest, int width, int heigth, int step, int channels) {

    // Compute global row and column indices
    int ren = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // Check for boundary conditions
    if (ren < 1 || col < 1 || ren >= heigth-1 || col >= width-1) return;

    // Sobel operator kernels
    int gx[3][3] = {{-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1}};
    int gy[3][3] = {{1, 2, 1}, {0, 0, 0}, {-1, -2, -1}};

    // Iterating over each color channel
    for (int c = 0; c < 3; c++) {
        int tmp_ren = 0;
        int tmp_col = 0;
        
        // Convolution operation
        for (int i = -1; i <= 1; i++) {
            for (int j = -1; j <= 1; j++) {
                int pixel = src[((ren + i) * step) + ((col + j) * channels) + c];
                tmp_ren += pixel * gx[i + 1][j + 1];
                tmp_col += pixel * gy[i + 1][j + 1];
            }
        }
        
        // Store result in destination image
        dest[(ren * step) + (col * channels) + c] = (unsigned char) sqrtf(tmp_col * tmp_col + tmp_ren * tmp_ren);
    }
}