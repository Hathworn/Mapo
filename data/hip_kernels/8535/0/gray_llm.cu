#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

typedef enum color {BLUE, GREEN, RED} Color;

// Optimized gray kernel function
__global__ void gray(unsigned char *src, unsigned char *dest, int width, int height, int step, int channels) {
    int ren = blockIdx.x;
    int col = threadIdx.x;
    
    // Check boundary conditions
    if (ren < height && col < width) {
        int pixel_index = (ren * step) + (col * channels);
        
        // Calculate gray value
        float r = (float) src[pixel_index + RED];
        float g = (float) src[pixel_index + GREEN];
        float b = (float) src[pixel_index + BLUE];
        unsigned char gray_value = (unsigned char) ((r + g + b) / 3);
        
        // Assign gray value to all color channels
        dest[pixel_index + RED] = gray_value;
        dest[pixel_index + GREEN] = gray_value;
        dest[pixel_index + BLUE] = gray_value;
    }
}