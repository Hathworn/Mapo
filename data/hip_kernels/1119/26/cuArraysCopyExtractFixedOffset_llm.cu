#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuArraysCopyExtractFixedOffset(const float *imageIn, const int inNX, const int inNY, float *imageOut, const int outNX, const int outNY, const int nImages, const int offsetX, const int offsetY)
{
    // Precompute constants
    const int outXY = outNX * outNY;
    const int inXY = inNX * inNY;

    // Calculate output indices
    int outx = threadIdx.x + blockDim.x * blockIdx.x;
    int outy = threadIdx.y + blockDim.y * blockIdx.y;
    int outz = blockIdx.z;

    // Check bounds first
    if (outx < outNX && outy < outNY)
    {
        // Compute flattened index for output and input arrays
        int idxOut = outz * outXY + outx * outNY + outy;
        int idxIn = outz * inXY + (outx + offsetX) * inNY + (outy + offsetY);

        // Copy data from input to output
        imageOut[idxOut] = imageIn[idxIn];
    }
}