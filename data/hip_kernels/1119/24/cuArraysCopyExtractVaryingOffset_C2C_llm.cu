#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuArraysCopyExtractVaryingOffset_C2C(const float2 *imageIn, const int inNX, const int inNY, float2 *imageOut, const int outNX, const int outNY, const int nImages, const int2 *offsets)
{
    // Calculate 1D index for threads for coalesced memory access
    int threadX = blockIdx.x * blockDim.x + threadIdx.x;
    int threadY = blockIdx.y * blockDim.y + threadIdx.y;
    int threadZ = blockIdx.z; // Each block in z-dimension handles different image

    // Ensure within output bounds
    if (threadX < outNX && threadY < outNY)
    {
        // Calculate input and output indices
        int idxOut = (threadZ * outNX + threadX) * outNY + threadY;
        int offsetX = offsets[threadZ].x;
        int offsetY = offsets[threadZ].y;
        int idxIn = (threadZ * inNX + threadX + offsetX) * inNY + threadY + offsetY;
        
        // Perform data copy
        imageOut[idxOut] = imageIn[idxIn];
    }
}