#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuArraysCopyExtract_C2R_FixedOffset(const float2 *imageIn, const int inNX, const int inNY, float *imageOut, const int outNX, const int outNY, const int nImages, const int offsetX, const int offsetY)
{
    // Calculate output indices
    int outx = threadIdx.x + blockDim.x * blockIdx.x;
    int outy = threadIdx.y + blockDim.y * blockIdx.y;

    // Check bounds and calculate indices for input and output
    if (outx < outNX && outy < outNY)
    {
        int idxOut = (blockIdx.z * outNX + outx) * outNY + outy;
        int idxIn = (blockIdx.z * inNX + (outx + offsetX)) * inNY + (outy + offsetY);
        
        // Perform copy operation
        imageOut[idxOut] = imageIn[idxIn].x;
    }
}