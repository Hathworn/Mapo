#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuArraysCopyToBatch_kernel(const float2 *imageIn, const int inNX, const int inNY, float2 *imageOut, const int outNX, const int outNY, const int nImagesX, const int nImagesY, const int strideX, const int strideY)
{
    // Calculate 1D global thread index for efficiency
    int outx = blockDim.x * blockIdx.x + threadIdx.x;
    int outy = blockDim.y * blockIdx.y + threadIdx.y;
    int idxImage = blockIdx.z;

    if (outx >= outNX || outy >= outNY || idxImage >= nImagesX * nImagesY) return;

    // Compute output index
    int idxOut = idxImage * outNX * outNY + outx * outNY + outy;

    // Pre-compute strides for image index calculation
    int idxImageX = idxImage / nImagesY;
    int idxImageY = idxImage % nImagesY;

    // Compute input index using precalculated strides
    int idxIn = (idxImageX * strideX + outx) * inNY + (idxImageY * strideY + outy);

    // Copy pixel from input to output
    imageOut[idxOut] = imageIn[idxIn];
}