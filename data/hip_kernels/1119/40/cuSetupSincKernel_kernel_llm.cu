#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuSetupSincKernel_kernel(float *r_filter_, const int i_filtercoef_, const float r_soff_, const float r_wgthgt_, const int i_weight_, const float r_soff_inverse_, const float r_beta_, const float r_decfactor_inverse_, const float r_relfiltlen_inverse_)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i >= i_filtercoef_) return; // Correct boundary condition

    // Pre-calculate constant expressions and reuse them
    const float cosine_arg = PI * r_soff_inverse_;
    const float r_weight_height_diff = 1.0f - r_wgthgt_;

    float r_wa = i - r_soff_;
    float r_wgt = r_weight_height_diff + r_wgthgt_ * cos(r_wa * cosine_arg); // Use pre-calculated cosine argument

    float r_s = r_wa * r_beta_ * r_decfactor_inverse_ * PI;
    float r_fct = (r_s != 0.0f) ? sin(r_s) / r_s : 1.0f; // Simplified conditional assignment

    if (i_weight_ == 1) {
        r_filter_[i] = r_fct * r_wgt;
    } else {
        r_filter_[i] = r_fct;
    }
}