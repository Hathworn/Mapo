#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuArraysCopyToBatchWithOffset_kernel(const float2 *imageIn, const int inNY, float2 *imageOut, const int outNX, const int outNY, const int nImages, const int *offsetX, const int *offsetY)
{
    // Calculate global thread coordinates
    int idxImage = blockIdx.z;
    int outx = threadIdx.x + blockDim.x * blockIdx.x;
    int outy = threadIdx.y + blockDim.y * blockIdx.y;

    // Early exit for threads outside the image range
    if(idxImage >= nImages || outx >= outNX || outy >= outNY) return;

    // Calculate flat indices for input and output arrays
    int idxOut = idxImage * outNX * outNY + outx * outNY + outy;
    int idxIn = (offsetX[idxImage] + outx) * inNY + offsetY[idxImage] + outy;

    // Copy image data from input to output
    imageOut[idxOut] = imageIn[idxIn];
}