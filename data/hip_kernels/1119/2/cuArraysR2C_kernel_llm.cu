#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* cuArraysPadding.cu
* Padding Utilities for oversampling
*/

// Padding zeros in the middle, move quads to corners
// for raw chunk data oversampling

__global__ void cuArraysR2C_kernel(float *image1, float2 *image2, int size)
{
    // Optimize access by calculating block index only once
    int idx =  blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure valid thread index before performing the operation
    if (idx < size)
    {
        float2 temp;
        temp.x = image1[idx];
        temp.y = 0.0f;
        image2[idx] = temp;  // Reduce memory access by writing results in one step
    }
}