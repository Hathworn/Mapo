#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuArraysCopyExtractVaryingOffset(const float *imageIn, const int inNX, const int inNY, float *imageOut, const int outNX, const int outNY, const int nImages, const int2 *offsets)
{
    // Calculate global thread index for 2D plane
    int outx = threadIdx.x + blockDim.x * blockIdx.x;
    int outy = threadIdx.y + blockDim.y * blockIdx.y;
    int idxImage = blockIdx.z;
    
    // Only proceed if thread's indexes are within bounds
    if(outx < outNX && outy < outNY)
    {
        // Compute input and output indices more efficiently
        int offsetX = offsets[idxImage].x;
        int offsetY = offsets[idxImage].y;

        int idxOut = (idxImage * outNX + outx) * outNY + outy;
        int idxIn = ((idxImage * inNX) + (outx + offsetX)) * inNY + (outy + offsetY);

        imageOut[idxOut] = imageIn[idxIn];
    }
}