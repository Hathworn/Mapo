#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuArraysCopyExtractVaryingOffsetCorr(const float *imageIn, const int inNX, const int inNY, float *imageOut, const int outNX, const int outNY, int *imageValid, const int nImages, const int2 *maxloc)
{
    // Calculate work item indices
    int idxImage = blockIdx.z;
    int outx = blockIdx.x * blockDim.x + threadIdx.x;
    int outy = blockIdx.y * blockDim.y + threadIdx.y;

    // Compute corresponding input indices
    int inx = outx + maxloc[idxImage].x - outNX / 2;
    int iny = outy + maxloc[idxImage].y - outNY / 2;

    if (outx < outNX && outy < outNY) {
        int idxOut = idxImage * outNX * outNY + outx * outNY + outy;
        
        // Check boundary and copy data
        if (inx >= 0 && iny >= 0 && inx < inNX && iny < inNY) {
            int idxIn = idxImage * inNX * inNY + inx * inNY + iny;
            imageOut[idxOut] = imageIn[idxIn];
            imageValid[idxOut] = 1;
        } else {
            imageOut[idxOut] = 0.0f;
            imageValid[idxOut] = 0;
        }
    }
}