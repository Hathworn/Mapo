#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuSubPixelOffset_kernel(const int2 *offsetInit, const int2 *offsetZoomIn, float2 *offsetFinal, const float OSratio, const float xoffset, const float yoffset, const int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Early exit for out-of-bounds threads
    if (idx >= size) return; 

    // Improved memory access pattern
    int2 offsetZ = offsetZoomIn[idx];
    int2 offsetI = offsetInit[idx];

    offsetFinal[idx].x = OSratio * offsetZ.x + offsetI.x - xoffset;
    offsetFinal[idx].y = OSratio * offsetZ.y + offsetI.y - yoffset;
}