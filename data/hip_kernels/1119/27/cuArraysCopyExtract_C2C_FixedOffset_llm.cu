#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuArraysCopyExtract_C2C_FixedOffset(const float2 *imageIn, const int inNX, const int inNY, float2 *imageOut, const int outNX, const int outNY, const int nImages, const int offsetX, const int offsetY)
{
    // Calculate global position
    int outx = threadIdx.x + blockDim.x * blockIdx.x;
    int outy = threadIdx.y + blockDim.y * blockIdx.y;
    int imgIdx = blockIdx.z;

    // Single condition check to ensure thread is within bounds
    if (outx < outNX && outy < outNY && imgIdx < nImages)
    {
        // Calculate indexes
        int idxOut = (imgIdx * outNX + outx) * outNY + outy;
        int idxIn = (imgIdx * inNX + outx + offsetX) * inNY + outy + offsetY;

        // Copy input to output
        imageOut[idxOut] = imageIn[idxIn];
    }
}