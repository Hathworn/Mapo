#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuSincInterpolation_kernel(const int nImages, const float * imagesIn, const int inNX, const int inNY, float * imagesOut, const int outNX, const int outNY, const float * r_filter_, const int i_covs_, const int i_decfactor_, const int i_intplength_, const int i_startX, const int i_startY, const int i_int_size)
{
    int idxImage = blockIdx.z;
    int idxX = threadIdx.x + blockDim.x * blockIdx.x;
    int idxY = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (idxImage >= nImages || idxX >= i_int_size || idxY >= i_int_size) return;

    int outx = idxX + i_startX;
    int outy = idxY + i_startY;
    int idxOut = idxImage * outNX * outNY + outx * outNY + outy;

    float r_xout = static_cast<float>(outx) / i_covs_;
    int i_xout = __float2int_rz(r_xout);  // Use fast intrinsic function
    float r_xfrac = r_xout - i_xout;
    int i_xfrac = __float2int_rz(r_xfrac * i_decfactor_);  // Use fast intrinsic function

    float r_yout = static_cast<float>(outy) / i_covs_;
    int i_yout = __float2int_rz(r_yout);  // Use fast intrinsic function
    float r_yfrac = r_yout - i_yout;
    int i_yfrac = __float2int_rz(r_yfrac * i_decfactor_);  // Use fast intrinsic function

    float intpData = 0.0f;
    float r_sincwgt = 0.0f;
    float r_sinc_coef;

    for (int i = 0; i < inNX; i++) {
        int i_xindex = i_xout - i + i_intplength_ / 2;
        i_xindex = (i_xindex + i_intplength_) % i_intplength_;  // Use modulo for index wrapping
        float r_xsinc_coef = r_filter_[i_xindex * i_decfactor_ + i_xfrac];

        for (int j = 0; j < inNY; j++) {
            int i_yindex = i_yout - j + i_intplength_ / 2;
            i_yindex = (i_yindex + i_intplength_) % i_intplength_;  // Use modulo for index wrapping
            float r_ysinc_coef = r_filter_[i_yindex * i_decfactor_ + i_yfrac];
            r_sinc_coef = r_xsinc_coef * r_ysinc_coef;
            r_sincwgt += r_sinc_coef;
            intpData += imagesIn[idxImage * inNX * inNY + i * inNY + j] * r_sinc_coef;
        }
    }
    imagesOut[idxOut] = intpData / r_sincwgt;
}