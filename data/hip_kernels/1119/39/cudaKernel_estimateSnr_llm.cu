#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaKernel_estimateSnr(const float* __restrict__ corrSum, const int* __restrict__ corrValidCount, const float* __restrict__ maxval, float* __restrict__ snrValue, const int size)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    
    if (idx < size) {
        // Use __restrict__ to potentially improve memory access patterns
        float mean = (corrSum[idx] - maxval[idx] * maxval[idx]) / (corrValidCount[idx] - 1);
        snrValue[idx] = maxval[idx] / mean;
    }
}