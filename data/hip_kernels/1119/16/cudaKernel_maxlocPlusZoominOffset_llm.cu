#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaKernel_maxlocPlusZoominOffset(float *offset, const int *padStart, const int *maxlocUpSample, const size_t nImages, float zoomInRatioX, float zoomInRatioY)
{
    int imageIndex = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index

    if (imageIndex < nImages)
    {
        int index = imageIndex * 2; // Optimize index calculation
        // Calculate and write offset using computed indices
        offset[index] = padStart[index] + maxlocUpSample[index] * zoomInRatioX;
        offset[index + 1] = padStart[index + 1] + maxlocUpSample[index + 1] * zoomInRatioY;
    }
}