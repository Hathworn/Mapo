#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* cuArraysPadding.cu
* Padding Utilities for oversampling
*/

// Padding zeros in the middle, move quads to corners
// For raw chunk data oversampling
// Tested
__global__ void cuArraysC2R_kernel(float2 *image1, float *image2, int size)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    // Use shared memory for optimized memory access
    __shared__ float2 sharedImage1[1024];

    if (idx < size)
    {
        // Load data from global to shared memory
        sharedImage1[threadIdx.x] = image1[idx];
        __syncthreads();

        // Store result from shared memory to global memory
        image2[idx] = sharedImage1[threadIdx.x].x;
    }
}