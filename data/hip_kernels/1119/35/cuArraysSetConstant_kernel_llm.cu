#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuArraysSetConstant_kernel(float *__restrict__ image, const int size, const float value)
{
    // Calculate global index
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // Use grid-stride loop to cover larger arrays
    for (; idx < size; idx += blockDim.x * gridDim.x)
    {
        image[idx] = value;
    }
}