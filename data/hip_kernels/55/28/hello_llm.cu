#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Load block index into shared memory for reduced redundancy
    __shared__ int blockIndex;
    
    if (threadIdx.x == 0) {
        blockIndex = blockIdx.x;
    }
    __syncthreads(); // Ensure shared memory is loaded before print

    printf("Hello world! I'm a thread in block %d\n", blockIndex);
}