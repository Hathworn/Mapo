#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT) {
    // Calculate global thread index
    int myId = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure within bounds before accessing memory
    if (myId < BIN_COUNT) { 
        int myItem = d_in[myId];
        int myBin = myItem % BIN_COUNT;
        // Use atomicAdd for thread-safe increments
        atomicAdd(&(d_bins[myBin]), 1);
    }
}