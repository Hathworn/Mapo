#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tonemap( float* __restrict__ d_x, float* __restrict__ d_y, float* __restrict__ d_log_Y, float* __restrict__ d_cdf_norm, float* __restrict__ d_r_new, float* __restrict__ d_g_new, float* __restrict__ d_b_new, float  min_log_Y, float  max_log_Y, float  log_Y_range, int    num_bins, int    num_pixels_y, int    num_pixels_x )
{
    // Calculate 1D image index
    int2 image_index_2d = make_int2( blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y );
    int image_index_1d = num_pixels_x * image_index_2d.y + image_index_2d.x;

    // Only process valid pixels
    if ( image_index_2d.x < num_pixels_x && image_index_2d.y < num_pixels_y )
    {
        float x = d_x[ image_index_1d ];
        float y = d_y[ image_index_1d ];
        float log_Y = d_log_Y[ image_index_1d ];
        
        // Compute histogram bin index
        int bin_index = min( num_bins - 1, static_cast<int>((num_bins * (log_Y - min_log_Y)) / log_Y_range) );
        float Y_new = d_cdf_norm[ bin_index ];

        // Calculate new color components
        float X_new = x * ( Y_new / y );
        float Z_new = ( 1.0f - x - y ) * ( Y_new / y );

        // Convert to RGB using matrix multiplication
        float r_new = X_new * 3.2406f + Y_new * -1.5372f + Z_new * -0.4986f;
        float g_new = X_new * -0.9689f + Y_new * 1.8758f + Z_new * 0.0415f;
        float b_new = X_new * 0.0557f + Y_new * -0.2040f + Z_new * 1.0570f;

        // Store results
        d_r_new[ image_index_1d ] = max(0.0f, min(1.0f, r_new));  // Clamp to [0, 1]
        d_g_new[ image_index_1d ] = max(0.0f, min(1.0f, g_new));  // Clamp to [0, 1]
        d_b_new[ image_index_1d ] = max(0.0f, min(1.0f, b_new));  // Clamp to [0, 1]
    }
}