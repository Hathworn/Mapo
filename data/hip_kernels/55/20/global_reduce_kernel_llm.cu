#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
    extern __shared__ float shared[]; // Allocate shared memory
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // Load input into shared memory
    shared[tid] = d_in[myId];
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            shared[tid] += shared[tid + s];
        }
        __syncthreads(); // Ensure all operations at this step are done
    }

    // Only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = shared[0];
    }
}