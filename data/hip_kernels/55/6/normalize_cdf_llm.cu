#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
{
    const float normalization_constant = 1.f / d_input_cdf[n - 1];

    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds and update output in a single if-statement
    if (global_index_1d < n)
    {
        d_output_cdf[global_index_1d] = d_input_cdf[global_index_1d] * normalization_constant;
    }
}