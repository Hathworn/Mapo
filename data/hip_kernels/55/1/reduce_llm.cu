#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int *s) {
    // Storing the input value to shared memory
    int idx = threadIdx.x;
    s[idx] = p;

    // Ensure all writes to shared memory are complete
    __syncthreads();

    // Warp-wide reduction using arithmetic operations
    if (idx < 16) s[idx] += s[idx + 16]; // Reduce within half-warp
    __syncthreads();
    if (idx < 8) s[idx] += s[idx + 8];
    __syncthreads();
    if (idx < 4) s[idx] += s[idx + 4];
    __syncthreads();
    if (idx < 2) s[idx] += s[idx + 2];
    __syncthreads();
    if (idx == 0) s[0] += s[1]; // Final reduction

    // Return the result stored in the first element of the shared memory
    return s[0];
}

__global__ void reduce(unsigned int *d_out_shared, const unsigned int *d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];

    // Calling shared reduce function
    unsigned int sr = shared_reduce(p, s);

    // Write the result from the first thread of the block to global memory
    if (t == 0) {
        *d_out_shared = sr;
    }
}