#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void remove_redness_from_coordinates( const unsigned int*  d_coordinates, unsigned char* d_r, unsigned char* d_b, unsigned char* d_g, unsigned char* d_r_output, int num_coordinates, int num_pixels_y, int num_pixels_x, int template_half_height, int template_half_width )
{
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;
    int imgSize = num_pixels_x * num_pixels_y;

    if (global_index_1d < num_coordinates)
    {
        unsigned int image_index_1d = d_coordinates[imgSize - global_index_1d - 1];
        ushort2 image_index_2d = make_ushort2(image_index_1d % num_pixels_x, image_index_1d / num_pixels_x);

        for (int y = image_index_2d.y - template_half_height; y <= image_index_2d.y + template_half_height; y++)
        {
            int clamped_y = min(num_pixels_y - 1, max(0, y)); // Clamp y coordinate
            for (int x = image_index_2d.x - template_half_width; x <= image_index_2d.x + template_half_width; x++)
            {
                int clamped_x = min(num_pixels_x - 1, max(0, x)); // Clamp x coordinate
                int image_offset_index_1d_clamped = clamped_y * num_pixels_x + clamped_x;

                unsigned int gb_average = (d_g[image_offset_index_1d_clamped] + d_b[image_offset_index_1d_clamped]) / 2; // Calculate GB average
                d_r_output[image_offset_index_1d_clamped] = (unsigned char)gb_average; // Update output array
            }
        }
    }
}