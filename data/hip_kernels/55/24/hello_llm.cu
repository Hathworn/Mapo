#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use more descriptive message and threadIdx.y to provide additional context
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}