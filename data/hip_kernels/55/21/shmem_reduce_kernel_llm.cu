#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shmem_reduce_kernel(float *d_out, const float *d_in) {
    // Shared memory allocated dynamically
    extern __shared__ float sdata[];

    int tid = threadIdx.x;
    int myId = tid + blockDim.x * blockIdx.x;

    // Load data into shared memory
    sdata[tid] = d_in[myId];
    __syncthreads(); // Ensure all data is loaded

    // Perform reduction using shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            // Reduction step
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // Synchronize threads at each step
    }

    // Write the result to global memory from thread 0
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}