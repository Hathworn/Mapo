#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void naive_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Calculate thread index for input element
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Use shared memory for bin increments to reduce global memory updates
    extern __shared__ int shared_bins[];
    int myBin = d_in[myId] % BIN_COUNT;

    // Initialize shared memory bins
    atomicAdd(&shared_bins[threadIdx.x], 0);
    __syncthreads();

    // Perform atomic addition on shared memory
    atomicAdd(&shared_bins[myBin], 1);
    __syncthreads();
    
    // Transfer results from shared to global memory
    if (threadIdx.x < BIN_COUNT) {
        atomicAdd(&d_bins[threadIdx.x], shared_bins[threadIdx.x]);
    }
}