#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void batcherBitonicMergesort64(float *d_out, const float *d_in) {
    // Optimized shared memory access and synchronization pattern
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();
    
    for (int stage = 0; stage <= 5; stage++) {
        int distance = 1 << stage;
        for (int substage = stage; substage >= 0; substage--) {
            int shift = 1 << substage;
            int index = 2 * tid - (tid & (shift - 1));
            // Efficient conditional check to avoid unnecessary computations
            if ((sdata[index] > sdata[index + shift]) == ((index & distance) == 0)) {
                float temp = sdata[index];
                sdata[index] = sdata[index + shift];
                sdata[index + shift] = temp;
            }
            __syncthreads();
        }
    }

    d_out[tid] = sdata[tid];
}