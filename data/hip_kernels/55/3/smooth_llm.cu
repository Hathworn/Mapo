#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth(float * v_new, const float * v) {
// Use 1D block and grid for efficient memory access and computation
int myIdx = blockIdx.x * blockDim.x + threadIdx.x;
int numThreads = blockDim.x * gridDim.x;

// Prevent out-of-bounds access
if (myIdx < numThreads) {
    // Clamp indices to avoid accessing out of array bounds
    int myLeftIdx = max(0, myIdx - 1);
    int myRightIdx = min(numThreads - 1, myIdx + 1);

    // Load elements once for computation
    float myElt = v[myIdx];
    float myLeftElt = v[myLeftIdx];
    float myRightElt = v[myRightIdx];

    // Store result in global memory
    v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
}
}