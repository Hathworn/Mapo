#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_copyRegion(unsigned char *dst, unsigned char *src, int stepDst, int stepSrc, int dst_width, int dst_height, int src_width, int src_height, int dst_xoffset, int dst_yoffset, int dst_widthToCrop, int dst_heightToCrop, int src_xoffset, int src_yoffset, int src_widthToCrop, int src_heightToCrop, int numChannel)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    // Compute destination and source indices
    int dst_col = col + dst_xoffset;
    int dst_row = row + dst_yoffset;
    int src_col = col + src_xoffset;
    int src_row = row + src_yoffset;

    // Check if pixel is within the region to be copied
    if (row < dst_heightToCrop && col < dst_widthToCrop && dst_col < dst_width && dst_row < dst_height)
    {
        // Efficiently copy data for different channels
        if (numChannel == 1)
        {
            dst[dst_row * stepDst + dst_col] = src[src_row * stepSrc + src_col];
        }
        else if (numChannel == 3)
        {
            int dst_step = dst_row * stepDst + dst_col * 3;
            int src_step = src_row * stepSrc + src_col * 3;
            dst[dst_step] = src[src_step];
            dst[dst_step + 1] = src[src_step + 1];
            dst[dst_step + 2] = src[src_step + 2];
        }
    }
}