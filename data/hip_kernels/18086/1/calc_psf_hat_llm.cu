#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calc_psf_hat(float* d_psf, float *d_psf_hat, int psf_rows, int psf_cols)
{
    // Use shared memory for faster access
    __shared__ float shared_psf[1024]; // Adjust size according to blockDim and psf size

    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int local_idx = threadIdx.y * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if(row < psf_rows && col < psf_cols)
    {
        shared_psf[local_idx] = d_psf[row * psf_cols + col];
    }
    __syncthreads();

    // Compute the result using shared memory
    if(row < psf_rows && col < psf_cols)
    {
        int index = (psf_rows - row - 1) * psf_cols + psf_cols - col - 1;
        d_psf_hat[index] = shared_psf[local_idx];
    }
}