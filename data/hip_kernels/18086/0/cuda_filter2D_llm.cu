#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to perform 2D filtering using shared memory
__global__ void cuda_filter2D(float *dst, float *src, float *kernel, int src_width, int src_height, int kernel_rows, int kernel_cols)
{
    // Calculate row and column indices
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    // Allocate shared memory for the input tile and kernel
    extern __shared__ float sharedMem[];
    float *sharedSrc = sharedMem;
    float *sharedKernel = &sharedMem[blockDim.x * blockDim.y];

    // Load kernel into shared memory
    for(int i = threadIdx.y; i < kernel_rows * kernel_cols; i += blockDim.y)
    {
        sharedKernel[i] = kernel[i];
    }
    __syncthreads();

    // Check if within bounds
    if(row < src_height && col < src_width)
    {
        float sum = 0;

        // Calculate start points
        int rowOffset = row - (kernel_rows - 1) / 2;
        int colOffset = col - (kernel_cols - 1) / 2;

        // Iterate over the kernel
        for(int i = 0; i < kernel_rows; i++)
        {
            for(int j = 0; j < kernel_cols; j++)
            {
                int curRow = rowOffset + i;
                int curCol = colOffset + j;

                // Bounds check
                if(curRow >= 0 && curRow < src_height && curCol >= 0 && curCol < src_width)
                {
                    sum += sharedKernel[i * kernel_cols + j] * src[curRow * src_width + curCol];
                }
            }
        }

        // Store the result with clamping
        dst[row * src_width + col] = fmaxf(0.0f, fminf(255.0f, sum));

        #ifdef debug
        printf("filter2D: dst[%d] = %f\n", row * src_width + col, dst[row * src_width + col]);
        #endif
    }
}