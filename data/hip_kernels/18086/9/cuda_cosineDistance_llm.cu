#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_cosineDistance(double *x, double *y, int64_t len, double *dot_product, double *norm_x, double *norm_y) 
{
    int64_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    int64_t cacheIdx = threadIdx.x;

    __shared__ double dot_cache[threadsPerBlock];
    __shared__ double norm_x_cache[threadsPerBlock];
    __shared__ double norm_y_cache[threadsPerBlock];

    double dot_tmp = 0;
    double norm_x_tmp = 0;
    double norm_y_tmp = 0;

    // Load data into cache
    while (idx < len) {
        dot_tmp += x[idx] * y[idx];
        norm_x_tmp += x[idx] * x[idx];
        norm_y_tmp += y[idx] * y[idx];
        idx += blockDim.x * gridDim.x;
    }
    
    dot_cache[cacheIdx] = dot_tmp;
    norm_x_cache[cacheIdx] = norm_x_tmp;
    norm_y_cache[cacheIdx] = norm_y_tmp;
    __syncthreads();

    // Reduce cache using warp shuffling
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        dot_tmp += __shfl_down_sync(0xFFFFFFFF, dot_tmp, offset);
        norm_x_tmp += __shfl_down_sync(0xFFFFFFFF, norm_x_tmp, offset);
        norm_y_tmp += __shfl_down_sync(0xFFFFFFFF, norm_y_tmp, offset);
    }

    if (cacheIdx % warpSize == 0) {
        atomicAdd(&dot_product[blockIdx.x], dot_tmp);
        atomicAdd(&norm_x[blockIdx.x], norm_x_tmp);
        atomicAdd(&norm_y[blockIdx.x], norm_y_tmp);
    }
}