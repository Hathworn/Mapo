#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_divide(float *dst, float *numerator, float *denominator, int width, int height)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Combine row and col into single index
    int totalThreads = gridDim.x * blockDim.x;  // Compute total number of threads

    for (int i = idx; i < width * height; i += totalThreads) {  // Loop to allow each thread to process multiple elements
        if (denominator[i] > 0.0000001) {
            dst[i] = numerator[i] / denominator[i];
        } else {
            dst[i] = 0;
        }
        // printf("dst[%d] = %f\n", i, dst[i]);
    }
}