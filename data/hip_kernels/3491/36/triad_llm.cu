#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void triad(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, float s)
{
    int gid = threadIdx.x + (blockIdx.x * blockDim.x);

    // Check if gid is within bounds before performing operations
    if (gid < blockDim.x * gridDim.x)
    {
        // Perform computation using registers to minimize memory access latency
        float a_val = A[gid];
        float b_val = B[gid];

        C[gid] = a_val + s * b_val;
    }
}