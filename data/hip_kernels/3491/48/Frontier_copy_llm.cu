#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Frontier_copy(unsigned int *frontier, unsigned int *frontier2, unsigned int *frontier_length) {
    // Obtain the linear thread index within the grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if index is within the bounds of the frontier_length
    if (idx < *frontier_length) {
        // Copy element from frontier to frontier2
        frontier2[idx] = frontier[idx];
    }
}