#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelUpdateBiases(float *nabla_b, float *biases, float eta, float mini_batch_size) {
    // Calculate rate once outside the loop
    float rate = eta / mini_batch_size;

    // Use blockIdx and blockDim for scalability
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < blockDim.x) { // Ensure within bounds
        biases[idx] -= rate * nabla_b[idx];
    }
}