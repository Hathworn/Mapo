#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readGlobalMemoryUnit(float *data, float *output, int size, int repeat)
{
    int gid = threadIdx.x + (blockDim.x * blockIdx.x);
    float sum = 0;
    int s = gid * 512;
    // Use loop unrolling and burst memory access
    for (int j = 0; j < repeat; ++j)
    {
        #pragma unroll 16
        for (int offset = 0; offset < 16; ++offset)
        {
            sum += data[(s + offset) & (size - 1)];
        }
        s = (s + 16) & (size - 1);
    }
    output[gid] = sum;
}