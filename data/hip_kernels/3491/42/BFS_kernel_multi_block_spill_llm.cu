#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BFS_kernel_multi_block_spill(volatile unsigned int *frontier, volatile unsigned int *frontier2, unsigned int frontier_len, volatile unsigned int *cost, volatile int *visited, unsigned int *edgeArray, unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, volatile unsigned int *frontier_length, const unsigned int max_local_mem)
{
    extern __shared__ unsigned int b_q[];

    __shared__ unsigned int b_q_length[1];
    __shared__ unsigned int b_offset[1];
    
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int lid = threadIdx.x;

    // Initialize shared memory variables
    if (lid == 0)
    {
        b_q_length[0] = 0;
        b_offset[0] = 0;
    }
    
    __syncthreads();

    if (tid < frontier_len)
    {
        unsigned int node_to_process = frontier[tid];
        visited[node_to_process] = 0;
        unsigned int offset = edgeArray[node_to_process];
        unsigned int next = edgeArray[node_to_process + 1];

        // Traverse neighbors
        while (offset < next)
        {
            unsigned int nid = edgeArrayAux[offset];
            unsigned int v = atomicMin(&cost[nid], cost[node_to_process] + 1);

            // Update frontier if required
            if (v > cost[node_to_process] + 1 && atomicExch(&visited[nid], 1) == 0)
            {
                unsigned int t = atomicAdd(&b_q_length[0], 1);

                // Use shared memory or spill to global memory
                if (t < max_local_mem)
                {
                    b_q[t] = nid;
                }
                else
                {
                    frontier2[atomicAdd((unsigned int *)frontier_length, 1)] = nid;
                }
            }
            offset++;
        }
    }
    
    __syncthreads();

    if (lid == 0)
    {
        if (b_q_length[0] > max_local_mem)
        {
            b_q_length[0] = max_local_mem;
        }
        b_offset[0] = atomicAdd((unsigned int *)frontier_length, b_q_length[0]);
    }
    
    __syncthreads();

    if (lid < b_q_length[0])
    {
        frontier2[lid + b_offset[0]] = b_q[lid];
    }
}