#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_one_block_spill(volatile unsigned int *frontier, unsigned int frontier_len, volatile unsigned int *cost, volatile int *visited, unsigned int *edgeArray, unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, volatile unsigned int *frontier_length, const unsigned int max_mem) {
    // Calculate the unique thread index
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure tid does not exceed frontier length
    if (tid < frontier_len) {
        unsigned int vertex = frontier[tid];
        
        // Traverse edges for the current vertex
        unsigned int start_edge = edgeArrayAux[vertex];
        unsigned int end_edge = edgeArrayAux[vertex + 1];

        for (unsigned int edge = start_edge; edge < end_edge; ++edge) {
            unsigned int neighbor = edgeArray[edge];

            // Check if neighbor has been visited
            if (!visited[neighbor]) {
                visited[neighbor] = 1;
                cost[neighbor] = cost[vertex] + 1;

                // Atomically add the neighbor vertex to the frontier
                unsigned int index = atomicAdd(frontier_length, 1);
                
                // Ensure memory safety with max_mem check
                if (index < max_mem) {
                    frontier[index] = neighbor;
                }
            }
        }
    }
}