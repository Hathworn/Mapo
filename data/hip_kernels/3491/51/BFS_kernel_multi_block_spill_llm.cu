#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel to improve memory access and reduce latency
__global__ void BFS_kernel_multi_block_spill(
    volatile unsigned int *frontier, volatile unsigned int *frontier2, unsigned int frontier_len, 
    volatile unsigned int *cost, volatile int *visited, 
    unsigned int *edgeArray, unsigned int *edgeArrayAux, 
    unsigned int numVertices, unsigned int numEdges, 
    volatile unsigned int *frontier_length, const unsigned int max_mem) 
{
    // Calculate thread ID within block
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for frequently accessed data
    extern __shared__ unsigned int shared_mem[];

    if (tid < frontier_len) {
        unsigned int vertex = frontier[tid];

        // Load edges into shared memory
        unsigned int start = edgeArray[vertex];
        unsigned int end = edgeArrayAux[vertex];
        
        for (unsigned int i = start; i < end; ++i) {
            unsigned int neighbor = edgeArray[i];
            
            if (!visited[neighbor]) {
                visited[neighbor] = 1;
                frontier2[atomicAdd(frontier_length, 1)] = neighbor;
            }
            __syncthreads(); // Ensure all threads in the block are synchronized
        }
    }

    // Ensure all memory operations are completed
    __threadfence_system();
}