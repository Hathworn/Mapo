#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFeedForward2(float *zs, float *biases, int b_off, float *activations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global index
    zs[idx] += biases[b_off + idx];  // Perform operation on global index
    activations[idx] = 1.0f / (1.0f + expf(-zs[idx]));  // Calculate activation
}