#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelUpdateWeights(float *nabla_w, float *weights, int tws, float eta, float mini_batch_size) {
    // Calculate rate outside the if statement, storing it in shared memory for efficient access.
    __shared__ float rate;
    if (threadIdx.x == 0) {
        rate = eta / mini_batch_size;
    }
    __syncthreads(); // Synchronize to ensure all threads have access to 'rate'.

    // Use a single calculation for the index.
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < tws) {
        // Use the pre-computed rate and index.
        weights[index] -= rate * nabla_w[index];
    }
}