#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelBackprop3a(float *delta_nabla_b, int b_off, int bound, int b_off_old, float *weights, int w_off_old) {
    // Use shared memory to cache weights
    extern __shared__ float shared_weights[];
    int thread_id = threadIdx.x; // Cache thread index
    int global_idx = b_off + thread_id; // Calculate global index
    
    delta_nabla_b[global_idx] = 0.0;
    for (int j = 0; j < bound; j++) {
        // Load weights to shared memory
        shared_weights[thread_id] = weights[w_off_old + (j * blockDim.x) + thread_id];
        __syncthreads(); // Ensure all weights are loaded
        
        // Compute delta_nabla_b using cached weights
        delta_nabla_b[global_idx] += delta_nabla_b[b_off_old + j] * shared_weights[thread_id];
        __syncthreads(); // Synchronize threads to ensure correct computation
    }
}