#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Frontier_copy( unsigned int *frontier, unsigned int *frontier2, unsigned int *frontier_length)
{
    // Calculate the global thread id
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform copy operation if within bounds
    if (tid < *frontier_length) {
        frontier[tid] = frontier2[tid];
    }

    // Use only the first thread in the grid to initialize global variables
    if (tid == 0) {
        g_mutex = 0;
        g_mutex2 = 0;
        *g_q_offsets = 0;
        *g_q_size = 0;
    }
}