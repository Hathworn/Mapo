#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readLocalMemory(const float *data, float *output, int size, int repeat)
{
    int gid = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    int localSize = blockDim.x;
    int grpid = blockIdx.x;
    int litems = 2048 / localSize;
    int goffset = localSize * grpid + tid * litems;
    int s = tid;
    __shared__ float lbuf[2048];

    // Efficient prefetching from global to shared memory
    int items_to_copy = min(litems, size - goffset);
    for (int j = 0; j < items_to_copy; ++j) {
        lbuf[tid * litems + j] = data[goffset + j];
    }
    
    __syncthreads();

    float sum = 0;
    #pragma unroll 16  // Unroll loop for improved ILP and pipeline utilization
    for (int j = 0; j < repeat; ++j) {
        #pragma unroll 16
        for (int i = 0; i < 16; ++i) {
            sum += lbuf[(s + i) & 2047];
        }
        s = (s + 16) & 2047; // Avoid expensive modulo operation
    }
    output[gid] = sum;
}