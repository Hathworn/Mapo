#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelBackprop3b(float *delta_nabla_b, int b_off, float *zs) {
    // Cache zs[threadIdx.x] value to reduce memory access
    float z = zs[threadIdx.x];
    // Calculate sigmoid(z) once and reuse for derivative
    float sigmoid_z = 1.0f / (1.0f + expf(-z));
    // Update delta_nabla_b using precomputed sigmoid value
    delta_nabla_b[b_off + threadIdx.x] *= sigmoid_z * (1.0f - sigmoid_z);
}