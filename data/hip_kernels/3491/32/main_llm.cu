#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "findRadixOffsets_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    uint2 *keys = NULL;
hipMalloc(&keys, XSIZE*YSIZE);
uint *counters = NULL;
hipMalloc(&counters, XSIZE*YSIZE);
uint *blockOffsets = NULL;
hipMalloc(&blockOffsets, XSIZE*YSIZE);
uint startbit = 1;
uint numElements = 1;
uint totalBlocks = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
findRadixOffsets<<<gridBlock, threadBlock>>>(keys,counters,blockOffsets,startbit,numElements,totalBlocks);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
findRadixOffsets<<<gridBlock, threadBlock>>>(keys,counters,blockOffsets,startbit,numElements,totalBlocks);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
findRadixOffsets<<<gridBlock, threadBlock>>>(keys,counters,blockOffsets,startbit,numElements,totalBlocks);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}