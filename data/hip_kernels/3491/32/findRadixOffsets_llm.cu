#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findRadixOffsets(uint2* keys, uint* counters, uint* blockOffsets, uint startbit, uint numElements, uint totalBlocks)
{
    __shared__ uint sStartPointers[16];
    extern __shared__ uint sRadix1[];

    uint groupId = blockIdx.x;
    uint localId = threadIdx.x;
    uint groupSize = blockDim.x;

    uint2 radix2 = keys[threadIdx.x + (blockIdx.x * blockDim.x)];

    // Store radix values in shared memory
    sRadix1[2 * localId]     = (radix2.x >> startbit) & 0xF;
    sRadix1[2 * localId + 1] = (radix2.y >> startbit) & 0xF;

    if (localId < 16)
    {
        sStartPointers[localId] = 0;
    }
    __syncthreads();

    // Use built-in warp functions for divergence detection
    if(localId < 2 * groupSize)
    {
        uint pos = (localId > 0 && sRadix1[localId] != sRadix1[localId - 1]) ? localId : 0;
        atomicMax(&sStartPointers[sRadix1[localId]], pos);
    }
    __syncthreads();

    if (localId < 16)
    {
        blockOffsets[groupId * 16 + localId] = sStartPointers[localId];
    }
    __syncthreads();

    // Calculate sizes of each block
    if(localId < 2 * groupSize)
    {
        uint prev = sRadix1[localId - 1];
        uint size = localId > 0 ? localId - sStartPointers[prev] : 0;
        if(prev != sRadix1[localId])
        {
            atomicMin(&sStartPointers[prev], size);
        }
    }

    if(localId == 2 * groupSize - 1)
    {
        uint last = sRadix1[localId];
        sStartPointers[last] = 2 * groupSize - sStartPointers[last];
    }
    __syncthreads();

    if(localId < 16)
    {
        counters[localId * totalBlocks + groupId] = sStartPointers[localId];
    }
}