#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelInitNablaW(float *nabla_w, int tws) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < tws) {
        nabla_w[idx] = 0.0; // Initialize nabla_w to zero
    }
}