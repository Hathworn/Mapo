#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_warp(unsigned int *levels, unsigned int *edgeArray, unsigned int *edgeArrayAux, int W_SZ, int CHUNK_SZ, unsigned int numVertices, int curr, int *flag) {
    // Calculate global and local thread indices
    unsigned int global_tid = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
    unsigned int local_tid = hipThreadIdx_x;

    // Ensure the current thread works over valid vertices
    if (global_tid < numVertices) {
        unsigned int level = levels[global_tid];

        // Iterate over neighbors using warp divergence optimization
        for (unsigned int i = edgeArray[global_tid]; i < edgeArrayAux[global_tid]; i++) {
            unsigned int neighbor = edgeArray[i];
            
            // Check and update level using atomic operations
            if (levels[neighbor] == 0xFFFFFFFF) { // Unvisited vertex
                levels[neighbor] = level + 1;
                *flag = 1;
            }
        }
    }
}