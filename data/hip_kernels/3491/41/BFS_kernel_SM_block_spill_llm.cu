#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define WARP_SIZE 32

__device__ void __gpu_sync(int blocks_to_synch) {
  __syncthreads();
  int tid_in_block = threadIdx.x;

  // Only thread 0 is used for synchronization
  if (tid_in_block == 0) {
    atomicAdd((int*)&g_mutex, 1);
    // Only when all blocks add 1 to g_mutex will g_mutex equal to blocks_to_synch
    while (g_mutex < blocks_to_synch);
  }
  __syncthreads();
}

__global__ void BFS_kernel_SM_block_spill(
    volatile unsigned int* frontier, volatile unsigned int* frontier2,
    unsigned int frontier_len, volatile unsigned int* cost,
    volatile int* visited, unsigned int* edgeArray, unsigned int* edgeArrayAux,
    unsigned int numVertices, unsigned int numEdges, volatile unsigned int* frontier_length,
    const unsigned int max_local_mem) {
  
  extern volatile __shared__ unsigned int b_q[];

  __shared__ unsigned int b_q_length;
  __shared__ unsigned int b_offset;

  unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
  unsigned int lid = threadIdx.x;

  int loop_index = 0;
  unsigned int l_mutex = g_mutex2;
  unsigned int f_len = frontier_len;

  while (1) {
    // Initialize the block queue length and warp queue offset
    if (lid == 0) {
      b_q_length = 0;
      b_offset = 0;
    }
    __syncthreads();

    // Processing nodes
    if (tid < f_len) {
      unsigned int node_to_process = (loop_index == 0) ? frontier[tid] : frontier2[tid];
      visited[node_to_process] = 0;  // Mark node as visited

      unsigned int offset = edgeArray[node_to_process];
      unsigned int next = edgeArray[node_to_process + 1];

      while (offset < next) {
        unsigned int nid = edgeArrayAux[offset];
        unsigned int v = atomicMin((unsigned int*)&cost[nid], cost[node_to_process] + 1);

        if (v > cost[node_to_process] + 1) {
          int is_in_frontier = atomicExch((int*)&visited[nid], 1);

          if (is_in_frontier == 0) {
            unsigned int t = atomicAdd((unsigned int*)&b_q_length, 1);
            if (t < max_local_mem) {
              b_q[t] = nid;
            }
            // Write to global memory if shared memory full
            else {
              int off = atomicAdd((unsigned int*)g_q_offsets, 1);
              if (loop_index == 0) {
                frontier2[off] = nid;
              } else {
                frontier[off] = nid;
              }
            }
          }
        }
        offset++;
      }
    }
    __syncthreads();

    if (lid == 0) {
      if (b_q_length > max_local_mem) {
        b_q_length = max_local_mem;
      }
      b_offset = atomicAdd((unsigned int*)g_q_offsets, b_q_length);
    }
    __syncthreads();

    l_mutex += gridDim.x;
    __gpu_sync(l_mutex);

    if (tid == 0) {
      g_q_size[0] = g_q_offsets[0];
      g_q_offsets[0] = 0;
    }

    if (lid < b_q_length) {
      if (loop_index == 0) {
        frontier2[lid + b_offset] = b_q[lid];
      } else {
        frontier[lid + b_offset] = b_q[lid];
      }
    }

    l_mutex += gridDim.x;
    __gpu_sync(l_mutex);

    if (g_q_size[0] < blockDim.x || g_q_size[0] > blockDim.x * gridDim.x) {
      break;
    }
    loop_index = (loop_index + 1) % 2;
    f_len = g_q_size[0];
  }

  if (loop_index == 0) {
    for (int i = tid; i < g_q_size[0]; i += blockDim.x * gridDim.x) {
      frontier[i] = frontier2[i];
    }
  }

  if (tid == 0) {
    frontier_length[0] = g_q_size[0];
  }
}