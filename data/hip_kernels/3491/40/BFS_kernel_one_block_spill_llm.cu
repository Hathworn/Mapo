#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_one_block_spill(volatile unsigned int *frontier, unsigned int frontier_len, volatile unsigned int *cost, volatile int *visited, unsigned int *edgeArray, unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, volatile unsigned int *frontier_length, const unsigned int max_local_mem)
{
    extern volatile __shared__ unsigned int s_mem[];
    
    // Block queues
    unsigned int *b_q = (unsigned int *)&s_mem[0];
    unsigned int *b_q2 = (unsigned int *)&s_mem[max_local_mem];
    
    volatile __shared__ unsigned int b_offset[1];
    volatile __shared__ unsigned int b_q_length[1];
    
    // Get the threadId
    unsigned int tid = threadIdx.x;

    // Load stable memory only once
    const unsigned int maxLM = max_local_mem; 

    // Copy frontier queue from global queue to local block queue
    if (tid < frontier_len) {
        b_q[tid] = frontier[tid];
    }

    unsigned int f_len = frontier_len;
    while (1) {
        // Initialize the block queue size to 0
        if (tid == 0) {
            b_q_length[0] = 0;
            b_offset[0] = 0;
        }
        __syncthreads();

        if (tid < f_len) {
            // Get the nodes to traverse from block queue
            unsigned int node_to_process = *(volatile unsigned int *)&b_q[tid];
            visited[node_to_process] = 0; // Remove from frontier

            // Get the offsets of the vertex in the edge list
            unsigned int offset = edgeArray[node_to_process];
            unsigned int next = edgeArray[node_to_process + 1];

            // Iterate through neighbors of the vertex
            while (offset < next) {
                unsigned int nid = edgeArrayAux[offset]; // Get neighbor
                unsigned int v = atomicMin((unsigned int *)&cost[nid], cost[node_to_process] + 1);

                // If cost is less than previously set add to frontier
                if (v > cost[node_to_process] + 1) {
                    int is_in_frontier = atomicExch((int *)&visited[nid], 1);
                    if (is_in_frontier == 0) { // If node already in frontier do nothing
                        unsigned int t = atomicAdd((unsigned int *)&b_q_length[0], 1);

                        // Add to local memory if not full
                        if (t < maxLM) {
                            b_q2[t] = nid;
                        } else {
                            int off = atomicAdd((unsigned int *)&b_offset[0], 1);
                            frontier[off] = nid; // Write to global memory if shared memory full
                        }
                    }
                }
                offset++;
            }
        }
        __syncthreads();

        if (tid < maxLM) {
            b_q[tid] = *(volatile unsigned int *)&b_q2[tid];
        }
        __syncthreads();

        // Traversal complete exit
        if (b_q_length[0] == 0) {
            if (tid == 0) {
                frontier_length[0] = 0;
            }
            return;
        }
        
        // If frontier exceeds one block in size copy warp queues to global frontier queue and exit
        else if (b_q_length[0] > blockDim.x || b_q_length[0] > maxLM) {
            if (tid < (b_q_length[0] - b_offset[0])) {
                frontier[b_offset[0] + tid] = *(volatile unsigned int *)&b_q[tid];
            }
            if (tid == 0) {
                frontier_length[0] = b_q_length[0];
            }
            return;
        }
        f_len = b_q_length[0];
        __syncthreads();
    }
}
```
