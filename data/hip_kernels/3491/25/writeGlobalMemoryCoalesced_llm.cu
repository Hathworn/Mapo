#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeGlobalMemoryCoalesced(float *output, int size, int repeat)
{
    int gid = threadIdx.x + (blockDim.x * blockIdx.x);
    int j = 0;

    // Using shared memory to reduce global memory access
    extern __shared__ float sharedOutput[];

    for (j = 0; j < repeat; ++j)
    {
        int s = gid;

        // Loop unrolling for improved memory access pattern
        #pragma unroll
        for (int offset = 0; offset < 524288; offset += 32768)
        {
            sharedOutput[threadIdx.x] = gid; // Coalesced write
            output[(s + offset) & (size - 1)] = sharedOutput[threadIdx.x];
        }

        s = (s + 524288) & (size - 1);
    }
}