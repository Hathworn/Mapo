#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelBackprop1(float *delta_nabla_w, int w_off, float *activations, float *delta_nabla_b, int b_off) {
    // Optimize by using shared memory for activations and delta_nabla_b
    extern __shared__ float shared_mem[];
    float* s_activations = shared_mem;
    float* s_delta_nabla_b = &shared_mem[blockDim.x];

    // Load data into shared memory
    int tid = threadIdx.x;
    s_activations[tid] = activations[tid];
    s_delta_nabla_b[blockIdx.x] = delta_nabla_b[b_off + blockIdx.x];
    __syncthreads();

    // Perform computation using shared memory
    delta_nabla_w[w_off + (blockIdx.x * blockDim.x) + threadIdx.x] = s_activations[tid] * s_delta_nabla_b[blockIdx.x];
}