#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_SM_block(volatile unsigned int *frontier, volatile unsigned int *frontier2, unsigned int frontier_len, volatile unsigned int *cost, volatile int *visited, unsigned int *edgeArray, unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, volatile unsigned int *frontier_length, unsigned int num_p_per_mp, unsigned int w_q_size) {
    // Calculate thread index within the block
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Ensure the thread does not exceed the number of vertices
    if (tid < frontier_len) {
        // Each thread works on a node from the frontier
        unsigned int node = frontier[tid];
        unsigned int start = edgeArray[node];
        unsigned int end = (node == numVertices - 1) ? numEdges : edgeArray[node + 1];
        
        // Explore neighbors of the node
        for (unsigned int edge = start; edge < end; ++edge) {
            unsigned int neighbor = edgeArrayAux[edge];
            
            // Check and set visited status for the neighbor
            if (atomicExch(&visited[neighbor], 1) == 0) {
                cost[neighbor] = cost[node] + 1;
                    
                // Append to the new frontier using atomic operation
                unsigned int index = atomicAdd((unsigned int*)frontier_length, 1);
                frontier2[index] = neighbor;
            }
        }
    }
}