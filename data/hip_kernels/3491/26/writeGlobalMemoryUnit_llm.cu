#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeGlobalMemoryUnit(float *output, int size, int repeat)
{
    int gid = threadIdx.x + (blockDim.x * blockIdx.x);
    int s = gid * 512;
    int max_index = size - 1;

    for (int j = 0 ; j < repeat ; ++j)
    {
        // Unroll loop and calculate address outside of the loop
        #pragma unroll
        for (int i = 0; i < 16; ++i)
        {
            output[(s + i) & max_index] = gid;
        }
        s = (s + 16) & max_index;
    }
}