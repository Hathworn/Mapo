#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readGlobalMemoryCoalesced(float *data, float *output, int size, int repeat)
{
    int gid = threadIdx.x + (blockDim.x * blockIdx.x);
    float sum = 0;
    int s = gid;

    #pragma unroll // Enable loop unrolling for potential performance gain
    for (int j = 0; j < repeat; ++j)
    {
        #pragma unroll // Unroll the accumulation loop
        for (int i = 0; i < 16; ++i)
        {
            sum += data[(s + i * 32768) & (size - 1)];
        }
        s = (s + 524288) & (size - 1); // Use a consistent update for 's'
    }
    output[gid] = sum;
}