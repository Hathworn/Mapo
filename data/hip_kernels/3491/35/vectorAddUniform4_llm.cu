#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAddUniform4(uint *d_vector, const uint *d_uniforms, const int n)
{
    __shared__ uint uni;

    // Load uniform value from global to shared memory
    if (threadIdx.x == 0)
    {
        uni = d_uniforms[blockIdx.x];
    }

    __syncthreads();

    unsigned int address = threadIdx.x + (blockIdx.x * blockDim.x * 4);

    // Use a single loop with a stride of blockDim.x * 4 to process multiple elements
    for (int i = 0; i < 4; i++, address += blockDim.x)
    {
        if (address < n)
        {
            d_vector[address] += uni;
        }
    }
}