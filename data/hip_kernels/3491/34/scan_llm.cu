#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ uint scanLocalMem(const uint val, uint* s_data) {
    int idx = threadIdx.x;
    s_data[idx] = 0.0f;
    __syncthreads();

    idx += blockDim.x;

// Reduced synchronizations for efficiency
    uint t;
    s_data[idx] = val;
    __syncthreads();
    if (idx >= 1) s_data[idx] += s_data[idx - 1];
    if (idx >= 2) s_data[idx] += s_data[idx - 2];
    if (idx >= 4) s_data[idx] += s_data[idx - 4];
    if (idx >= 8) s_data[idx] += s_data[idx - 8];
    if (idx >= 16) s_data[idx] += s_data[idx - 16];
    if (idx >= 32) s_data[idx] += s_data[idx - 32];
    if (idx >= 64) s_data[idx] += s_data[idx - 64];
    if (idx >= 128) s_data[idx] += s_data[idx - 128];

    return s_data[idx-1];
}

__global__ void scan(uint *g_odata, uint* g_idata, uint* g_blockSums, const int n, const bool fullBlock, const bool storeSum) {
    __shared__ uint s_data[512];

    uint4 tempData;
    uint4 threadScanT;
    uint res;
    uint4* inData  = (uint4*) g_idata;
    const int gid = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int tid = threadIdx.x;
    const int i = gid * 4;

    if (fullBlock || i + 3 < n) {
        tempData = inData[gid];
        threadScanT.x = tempData.x;
        threadScanT.y = tempData.y + threadScanT.x;
        threadScanT.z = tempData.z + threadScanT.y;
        threadScanT.w = tempData.w + threadScanT.z;
        res = threadScanT.w;
    } else {
        threadScanT.x = (i < n) ? g_idata[i] : 0.0f;
        threadScanT.y = ((i+1 < n) ? g_idata[i+1] : 0.0f) + threadScanT.x;
        threadScanT.z = ((i+2 < n) ? g_idata[i+2] : 0.0f) + threadScanT.y;
        threadScanT.w = ((i+3 < n) ? g_idata[i+3] : 0.0f) + threadScanT.z;
        res = threadScanT.w;
    }

    res = scanLocalMem(res, s_data);
    __syncthreads();

    if (storeSum && tid == blockDim.x-1) {
        g_blockSums[blockIdx.x] = res + threadScanT.w;
    }

    uint4* outData = (uint4*) g_odata;
    tempData.x = res;
    tempData.y = res + threadScanT.x;
    tempData.z = res + threadScanT.y;
    tempData.w = res + threadScanT.z;

    if (fullBlock || i + 3 < n) {
        outData[gid] = tempData;
    } else {
        if (i < n) {
            g_odata[i] = tempData.x;
            if ((i+1) < n) { g_odata[i+1] = tempData.y; }
            if ((i+2) < n) { g_odata[i+2] = tempData.z; }
        }
    }
}