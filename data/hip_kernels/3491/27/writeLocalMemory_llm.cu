#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeLocalMemory(float *output, int size, int repeat)
{
    int gid = threadIdx.x + (blockDim.x * blockIdx.x);
    int tid = threadIdx.x, localSize = blockDim.x;
    int s = tid;
    __shared__ float lbuf[2048];
    
    // Optimize by unrolling loop with compiler directive
    #pragma unroll
    for (int j = 0; j < repeat; ++j)
    {
        #pragma unroll 16
        for (int k = 0; k < 16; ++k)
        {
            lbuf[(s + k) & 2047] = gid;
        }
        s = (s + 16) & 2047;
    }
    __syncthreads();
    
    int litems = 2048 / localSize; // Move this declaration closer to usage
    
    // Minimize redundant global memory writes by incremental accesses
    int offset = tid;
    for (int j = 0; j < litems; ++j, offset += localSize)
    {
        output[gid] = lbuf[offset & 2047];
    }
}