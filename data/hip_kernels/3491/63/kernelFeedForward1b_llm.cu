#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFeedForward1b(float *zs, int bound, float *weights, int w_off, float *activations) {
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    float sum = 0.0f; // Accumulator for the output

    // Loop unrolling for improved performance
    for (int i = 0; i < bound; i += 4) {
        sum += weights[w_off + (threadIdx.x * bound) + i] * activations[(blockIdx.x * bound) + i];
        if (i + 1 < bound) sum += weights[w_off + (threadIdx.x * bound) + i + 1] * activations[(blockIdx.x * bound) + i + 1];
        if (i + 2 < bound) sum += weights[w_off + (threadIdx.x * bound) + i + 2] * activations[(blockIdx.x * bound) + i + 2];
        if (i + 3 < bound) sum += weights[w_off + (threadIdx.x * bound) + i + 3] * activations[(blockIdx.x * bound) + i + 3];
    }

    zs[idx] = sum; // Save result back to global memory
}