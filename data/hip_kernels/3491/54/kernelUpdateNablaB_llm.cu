#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelUpdateNablaB(float *nabla_b, float *delta_nabla_b) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global index
    nabla_b[idx] += delta_nabla_b[idx];               // Use global index for memory access
}