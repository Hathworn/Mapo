#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFeedForward3(float *zs, float *biases, int b_off, float *activations) {
    // Calculate global thread index
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Use a single array access expression
    float z_val = zs[idx] + biases[b_off + threadIdx.x];
    
    // Write back the computation
    zs[idx] = z_val;
    
    // Optimized activation computation
    activations[idx] = 1.0 / (1.0 + expf(-z_val));
}