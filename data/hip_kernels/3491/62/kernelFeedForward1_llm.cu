#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFeedForward1(float *zs, int bound2, float *weights, int w_off, float *activations1) {
    // Calculate the thread ID
    int tid = threadIdx.x;

    // Initialize the result for this thread
    float sum = 0.0;

    // Use a register to accumulate the result and avoid global memory access in each iteration
    for (int i = 0; i < bound2; i++) {
        sum += weights[w_off + (tid * bound2) + i] * activations1[i];
    }

    // Write the result to global memory only once
    zs[tid] = sum;
}