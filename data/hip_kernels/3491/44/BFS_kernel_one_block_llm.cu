#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_one_block(volatile unsigned int *frontier, unsigned int frontier_len, volatile unsigned int *cost, volatile int *visited, unsigned int *edgeArray, unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, volatile unsigned int *frontier_length, unsigned int num_p_per_mp, unsigned int w_q_size) {
    // Calculate thread index
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure thread index is within frontier length
    if (idx < frontier_len) {

        // Load frontier data to register (coalesced memory access)
        unsigned int vertex = frontier[idx];

        // Process only unvisited vertices
        if (visited[vertex] == 0) {

            // Iterate over neighbors
            unsigned int start_edge = edgeArrayAux[vertex];
            unsigned int end_edge = edgeArrayAux[vertex + 1];
            
            for (unsigned int edge = start_edge; edge < end_edge; ++edge) {
                unsigned int neighbor = edgeArray[edge];
                
                // Update cost and mark as visited
                if (atomicMin(&cost[neighbor], cost[vertex] + 1)) {
                    atomicExch(&visited[neighbor], 1);
                }
            }
        }

        // Update frontier length atomically (to avoid race conditions)
        if (visited[vertex] == 1) {
            atomicAdd(frontier_length, 1);
        }
    }
}