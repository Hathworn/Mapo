#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelUpdateNablaW(float *nabla_w, float *delta_nabla_w, int tws) {
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the index is within bounds and perform addition in one step
    if (idx < tws) {
        nabla_w[idx] += delta_nabla_w[idx];
    }
}