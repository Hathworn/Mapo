#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reorderData(uint startbit, uint *outKeys, uint *outValues, uint2 *keys, uint2 *values, uint *blockOffsets, uint *offsets, uint *sizes, uint totalBlocks)
{
    uint GROUP_SIZE = blockDim.x;
    __shared__ uint2 sKeys2[256];
    __shared__ uint2 sValues2[256];
    __shared__ uint sOffsets[16];
    __shared__ uint sBlockOffsets[16];
    uint* sKeys1   = (uint*) sKeys2;
    uint* sValues1 = (uint*) sValues2;

    uint blockId = blockIdx.x;
    uint i = blockId * blockDim.x + threadIdx.x;

    sKeys2[threadIdx.x]   = keys[i];
    sValues2[threadIdx.x] = values[i];

    if(threadIdx.x < 16)
    {
        sOffsets[threadIdx.x]      = offsets[threadIdx.x * totalBlocks + blockId];
        sBlockOffsets[threadIdx.x] = blockOffsets[blockId * 16 + threadIdx.x];
    }
    __syncthreads();

    // Reduce repeated calculations of radix and globalOffset
    for (int step = 0; step < 2; ++step) {
        uint idx = threadIdx.x + step * GROUP_SIZE;
        uint radix = (sKeys1[idx] >> startbit) & 0xF;
        uint globalOffset = sOffsets[radix] + idx - sBlockOffsets[radix];
        
        outKeys[globalOffset] = sKeys1[idx];
        outValues[globalOffset] = sValues1[idx];
    }
}