#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "reorderData_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    uint startbit = 1;
uint *outKeys = NULL;
hipMalloc(&outKeys, XSIZE*YSIZE);
uint *outValues = NULL;
hipMalloc(&outValues, XSIZE*YSIZE);
uint2 *keys = NULL;
hipMalloc(&keys, XSIZE*YSIZE);
uint2 *values = NULL;
hipMalloc(&values, XSIZE*YSIZE);
uint *blockOffsets = NULL;
hipMalloc(&blockOffsets, XSIZE*YSIZE);
uint *offsets = NULL;
hipMalloc(&offsets, XSIZE*YSIZE);
uint *sizes = NULL;
hipMalloc(&sizes, XSIZE*YSIZE);
uint totalBlocks = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
reorderData<<<gridBlock, threadBlock>>>(startbit,outKeys,outValues,keys,values,blockOffsets,offsets,sizes,totalBlocks);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
reorderData<<<gridBlock, threadBlock>>>(startbit,outKeys,outValues,keys,values,blockOffsets,offsets,sizes,totalBlocks);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
reorderData<<<gridBlock, threadBlock>>>(startbit,outKeys,outValues,keys,values,blockOffsets,offsets,sizes,totalBlocks);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}