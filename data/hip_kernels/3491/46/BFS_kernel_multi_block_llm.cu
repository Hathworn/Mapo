#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Simplified kernel function name
__global__ void BFS_optimized_kernel(
    volatile unsigned int *frontier,
    volatile unsigned int *frontier2,
    unsigned int frontier_len,
    volatile unsigned int *cost,
    volatile int *visited,
    unsigned int *edgeArray,
    unsigned int *edgeArrayAux,
    unsigned int numVertices,
    unsigned int numEdges,
    volatile unsigned int *frontier_length,
    unsigned int NUM_P_PER_MP,
    unsigned int W_Q_SIZE) 
{
    // Obtain thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure threads do not exceed frontier length
    if (tid < frontier_len) {
        unsigned int vertex = frontier[tid];
        
        // Process edges of the current vertex
        unsigned int start_edge = edgeArrayAux[vertex];
        unsigned int end_edge = edgeArrayAux[vertex + 1];
        
        for (unsigned int edge = start_edge; edge < end_edge; ++edge) {
            unsigned int neighbor = edgeArray[edge];
            
            // Check if the neighbor has already been visited
            if (visited[neighbor] == 0) {
                visited[neighbor] = 1;
                frontier2[atomicAdd(frontier_length, 1)] = neighbor;
                
                // Update cost
                cost[neighbor] = cost[vertex] + 1;
            }
        }
    }
}