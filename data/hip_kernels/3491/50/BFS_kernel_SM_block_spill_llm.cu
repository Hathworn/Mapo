#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_SM_block_spill(volatile unsigned int *frontier, 
                                          volatile unsigned int *frontier2, 
                                          unsigned int frontier_len, 
                                          volatile unsigned int *cost, 
                                          volatile int *visited, 
                                          unsigned int *edgeArray, 
                                          unsigned int *edgeArrayAux, 
                                          unsigned int numVertices, 
                                          unsigned int numEdges, 
                                          volatile unsigned int *frontier_length, 
                                          const unsigned int max_mem) {
    // Calculate thread and block index
    unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int stride = blockDim.x * gridDim.x;

    // Process each frontier in stride of available threads
    for (unsigned int index = tid; index < frontier_len; index += stride) {
        unsigned int node = frontier[index];

        // If node was not visited
        if (!visited[node]) {
            visited[node] = 1; // Mark as visited

            // Iterate over all edges of the node
            unsigned int start = edgeArray[node];
            unsigned int end = (node + 1 < numVertices) ? edgeArray[node + 1] : numEdges;

            // Loop through edges
            for (unsigned int j = start; j < end; ++j) {
                unsigned int neighbor = edgeArrayAux[j];

                // If neighbor wasn't visited, update cost and add to frontier2
                if (!visited[neighbor]) {
                    cost[neighbor] = cost[node] + 1;
                    unsigned int old_len = atomicAdd(frontier_length, 1);
                    if (old_len < max_mem) {
                        frontier2[old_len] = neighbor;
                    }
                }
            }
        }
    }
}