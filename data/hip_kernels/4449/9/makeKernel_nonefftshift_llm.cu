#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void makeKernel_nonefftshift(float* KernelPhase, int row, int column, float* ImgProperties) {
    // Cache ImgProperties in shared memory for faster access
    __shared__ float sharedImgProps[5];
    if (threadIdx.x < 5) {
        sharedImgProps[threadIdx.x] = ImgProperties[threadIdx.x];
    }
    __syncthreads();

    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    // Use registers for frequently accessed variables
    float pixSize = sharedImgProps[0];
    float MagX = sharedImgProps[1];
    float nmed = sharedImgProps[2];
    float lambda = sharedImgProps[3];
    float MagXscaling = 1.0f / sharedImgProps[4];
    float pixdxInv = MagX / pixSize * MagXscaling; // Magnification/pixSize
    float km = nmed / lambda; // nmed / lambda

    for (int i = threadID; i < row * column; i += numThreads) {
        int dx = i % row;
        int dy = i / row;
        
        // Use bitwise operations for conditional assignments
        dx -= (dx >= row / 2) * row;
        dy -= (dy >= row / 2) * row;

        float kdx = dx * pixdxInv;
        float kdy = dy * pixdxInv;
        float temp = km * km - kdx * kdx - kdy * kdy;
        KernelPhase[i] = (temp >= 0) ? (sqrtf(temp) - km) : 0;
    }
}