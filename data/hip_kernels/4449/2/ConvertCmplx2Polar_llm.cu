#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access by using shared memory
__global__ void ConvertCmplx2Polar(float* inRe, float* inIm, float* mag, float* phase, int size) {
    __shared__ float sharedRe[BLOCK_SIZE];
    __shared__ float sharedIm[BLOCK_SIZE];
    
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  
    for (int i = threadID; i < size; i += blockDim.x * gridDim.x) {
        // Load data into shared memory
        sharedRe[threadIdx.x] = inRe[i];
        sharedIm[threadIdx.x] = inIm[i];
        
        __syncthreads();
        
        // Calculate polar coordinates
        phase[i] = atan2f(sharedIm[threadIdx.x], sharedRe[threadIdx.x]);
        mag[i] = cmagf(sharedIm[threadIdx.x], sharedRe[threadIdx.x]);
        
        __syncthreads();
    }
}

__device__ static __inline__ float cmagf(float x, float y) {
    float a, b, v, w, t;
    a = fabsf(x);
    b = fabsf(y);
    if (a > b) {
        v = a;
        w = b;
    } else {
        v = b;
        w = a;
    }
    t = w / v;
    t = 1.0f + t * t;
    t = v * sqrtf(t);
    if ((v == 0.0f) || (v > 3.402823466e38f) || (w > 3.402823466e38f)) {
        t = v + w;
    }
    return t;
}