#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ScaleMatrix_Kernel(float *d_a, float alpha, int arraySize)
{
    // Compute global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check bounds
    if (index < arraySize) {
        // Scale element
        d_a[index] = alpha * d_a[index];
    }
}