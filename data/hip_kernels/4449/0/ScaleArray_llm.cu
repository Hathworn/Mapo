#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ScaleArray(float *d_a, float alpha, int arraySize)
{
    // Calculate the global thread ID
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize by using a single loop stride within bounds
    for (int i = threadID; i < arraySize; i += blockDim.x * gridDim.x)
    {
        d_a[i] *= alpha; // Simplified multiplication
    }
}