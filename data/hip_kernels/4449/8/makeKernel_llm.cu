#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void makeKernel(float* KernelPhase, int row, int column, float* ImgProperties, float MagXscaling) {
    // Precompute constant values once
    float MagX = ImgProperties[1];
    float pixSize = ImgProperties[0];
    float nm = ImgProperties[2];
    float lambda = ImgProperties[3];

    float pixdxInv = MagX / pixSize * MagXscaling;
    float km = nm / lambda;

    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop where possible and reduce thread divergence
    for (int i = threadID; i < row * column; i += numThreads) {
        int dx = i % row;
        int dy = i / row;

        // Calculate kdx and kdy once per loop iteration
        float kdx = (dx - row / 2.0f) * pixdxInv;
        float kdy = (dy - row / 2.0f) * pixdxInv;
        float temp = km * km - kdx * kdx - kdy * kdy;

        // Use ternary operator for conditionally setting KernelPhase
        KernelPhase[i] = (temp >= 0) ? (sqrtf(temp) - km) : 0;
    }
}