#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void makeKernelPhase(float* KernelPhase, int row, int column, float* ImgProperties) {

    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    const float pixdxInv = ImgProperties[1] / ImgProperties[0]; // Magnification/pixSize
    const float km = ImgProperties[2] / ImgProperties[3]; // nm / lambda

    for (int i = threadID; i < row * column; i += blockDim.x * gridDim.x) {
        int dx = i % row;
        int dy = i / row;

        dx = dx - (row >> 1); // Use bitwise shift for division
        dy = dy - (row >> 1); // Use bitwise shift for division

        float kdx = float(dx) * pixdxInv / row;
        float kdy = float(dy) * pixdxInv / row;
        float temp = km * km - kdx * kdx - kdy * kdy;

        KernelPhase[i] = (temp >= 0) ? (sqrtf(temp) - km) : 0;
    }

}