#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tonemap( float* d_x, float* d_y, float* d_log_Y, float* d_cdf_norm, float* d_r_new, float* d_g_new, float* d_b_new, float  min_log_Y, float  max_log_Y, float  log_Y_range, int    num_bins, int    num_pixels_y, int    num_pixels_x )
{
    int nx = num_pixels_x;
    int ny = num_pixels_y;
    
    // Calculate 1D flattened index using block and thread indices
    int2 image_index_2d = make_int2( ( blockIdx.x * blockDim.x ) + threadIdx.x, ( blockIdx.y * blockDim.y ) + threadIdx.y );
    int image_index_1d = ( nx * image_index_2d.y ) + image_index_2d.x;

    if (image_index_2d.x < nx && image_index_2d.y < ny)
    {
        // Fetch input data from device memory
        float x = d_x[image_index_1d];
        float y = d_y[image_index_1d];
        float log_Y = d_log_Y[image_index_1d];

        // Compute bin index with possible pre-calculation for efficiency
        int bin_index = min(num_bins - 1, int( (num_bins * (log_Y - min_log_Y)) / log_Y_range ));
        float Y_new = d_cdf_norm[bin_index];

        // Pre-compute reciprocal of y to avoid multiple divisions
        float inv_y = 1.0f / y;

        // Calculate color components with matrix multiplication
        float X_new = x * (Y_new * inv_y);
        float Z_new = (1.0f - x - y) * (Y_new * inv_y);

        // Use more efficient arithmetic operations by restructuring equations
        float r_new = fmaf(X_new, 3.2406f, fmaf(Y_new, -1.5372f, Z_new * -0.4986f));
        float g_new = fmaf(X_new, -0.9689f, fmaf(Y_new, 1.8758f, Z_new * 0.0415f));
        float b_new = fmaf(X_new, 0.0557f, fmaf(Y_new, -0.2040f, Z_new * 1.0570f));

        // Store computed colors in the output arrays
        d_r_new[image_index_1d] = r_new;
        d_g_new[image_index_1d] = g_new;
        d_b_new[image_index_1d] = b_new;
    }
}