#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalize_cdf( unsigned int* d_input_cdf, float* d_output_cdf, int n )
{
    // Calculate constant only once per block instead of per thread
    __shared__ float normalization_constant;
    if (threadIdx.x == 0)
    {
        normalization_constant = 1.f / d_input_cdf[n - 1];
    }
    __syncthreads(); // Ensure all threads in the block have the updated constant

    int global_index_1d = ( blockIdx.x * blockDim.x ) + threadIdx.x;

    if ( global_index_1d < n )
    {
        // Use shared normalization constant instead of recalculating
        d_output_cdf[ global_index_1d ] = d_input_cdf[ global_index_1d ] * normalization_constant;
    }
}