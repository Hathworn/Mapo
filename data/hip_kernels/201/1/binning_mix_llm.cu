#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Constants */

#define threads 256 /* It's the number of threads we are going to use per block on the GPU */

/* Kernels */

__global__ void binning_mix(float *xd_real, float *yd_real, float *zd_real, float *xd_sim, float *yd_sim, float *zd_sim, float *ZY, int lines_number_1, int lines_number_2, int points_per_degree, int number_of_degrees)
{
    /* Shared memory for storing temporary results */
    __shared__ float temp[threads];

    /* Each thread initializes its shared memory slot to zero */
    temp[threadIdx.x] = 0;
    __syncthreads(); /* Ensure all threads have initialized their shared memory */

    /* Iterate over the first set of data */
    for (int i = 0; i < lines_number_1; i++)
    {
        float x = xd_real[i];
        float y = yd_real[i];
        float z = zd_real[i];

        /* Compute using block-level parallelism */
        for (int dim_idx = blockIdx.x * blockDim.x + threadIdx.x;
             dim_idx < lines_number_2;
             dim_idx += blockDim.x * gridDim.x)
        {
            float xx = xd_sim[dim_idx];
            float yy = yd_sim[dim_idx];
            float zz = zd_sim[dim_idx];

            /* Calculate the dot product and convert to angle */
            float angle = x * xx + y * yy + z * zz;
            angle = fminf(angle, 1.0);
            angle = acosf(angle) * 180.0 / M_PI;

            /* Count pairs if angle < number_of_degrees */
            if (angle < number_of_degrees)
            {
                atomicAdd(&temp[int(angle * points_per_degree)], 1.0f);
            }
        }
    }

    /* Copy results from shared to global memory */
    atomicAdd(&ZY[threadIdx.x], temp[threadIdx.x]);
}