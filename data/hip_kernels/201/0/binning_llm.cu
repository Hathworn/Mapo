#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Constants */
#define threads 256 /* It's the number of threads we are going to use per block on the GPU */

/* Kernels */
__global__ void binning(float *xd, float *yd, float *zd, float *ZZ, int number_lines, int points_per_degree, int number_of_degrees)
{
    /* Shared memory for temporary accumulation */
    __shared__ float temp[threads];
    int index = threadIdx.x;
    temp[index] = 0.0f;

    /* Start counting pairs */
    for (int i = blockIdx.x; i < number_lines; i += gridDim.x)
    {
        float x = xd[i];
        float y = yd[i];
        float z = zd[i];

        /* Calculate dot products */
        for (int dim_idx = threadIdx.x; dim_idx < number_lines; dim_idx += blockDim.x)
        {
            float xx = xd[dim_idx];
            float yy = yd[dim_idx];
            float zz = zd[dim_idx];

            /* Compute the angle */
            float angle = fminf(x * xx + y * yy + z * zz, 1.0f);
            angle = acosf(angle) * 180.0f / M_PI;

            /* Count pairs */
            if (angle < number_of_degrees)
            {
                atomicAdd(&temp[int(angle * points_per_degree)], 1.0f);
            }
        }
    }

    __syncthreads();

    /* Accumulate results from shared to global memory */
    if (index < number_of_degrees * points_per_degree)
    {
        atomicAdd(&ZZ[index], temp[index]);
    }
}