#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized mapPrefixSumToPrisms kernel
__global__ void mapPrefixSumToPrisms(const unsigned numberOfPrisms, const unsigned raysPerSample, const unsigned reflectionSlices, const unsigned* raysPerPrism, const unsigned* prefixSum, unsigned *indicesOfPrisms, unsigned *numberOfReflections) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure valid thread id
    if (id >= numberOfPrisms * reflectionSlices) return;

    // Pre-calculate values used in the loop to reduce repetitive calculations
    const unsigned count = raysPerPrism[id];
    const unsigned startingPosition = prefixSum[id];
    const unsigned reflection_i = id / numberOfPrisms; 
    const unsigned prism_i = id % numberOfPrisms;

    // Use pointer arithmetic for more efficient memory access
    unsigned *indicesPtr = &indicesOfPrisms[startingPosition];
    unsigned *reflectionsPtr = &numberOfReflections[startingPosition];

    // Loop unrolling might help further if 'count' is small and predictable
    for (unsigned i = 0; i < count; ++i) {
        indicesPtr[i] = prism_i;
        reflectionsPtr[i] = reflection_i;
    }
}