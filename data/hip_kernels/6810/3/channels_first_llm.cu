#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void channels_first(float* input, float* rinput, int channels, int height, int width, int pad_size)
{
    // Calculate indices for batch, height, width, and channel offset
    int n = blockIdx.x;
    int y = blockIdx.y;
    int x = blockIdx.z;
    int ch_off = threadIdx.x;

    // Precompute dimension products for indexing
    int dimcyx = channels * height * width;
    int dimyx = height * width;
    int p_dimx = (width + 2 * pad_size);
    int p_dimy = (height + 2 * pad_size);
    int p_dimyxc = channels * p_dimy * p_dimx;
    int p_dimxc = p_dimx * channels;

    // Loop over channels with stride to handle multiple channels per thread
    for (int c = ch_off; c < channels; c += blockDim.x) { // Use blockDim.x for thread stride
        // Access the input array and store the value in the padded output array
        float value = input[n * dimcyx + c * dimyx + y * width + x];
        rinput[n * p_dimyxc + (y + pad_size) * p_dimxc + (x + pad_size) * channels + c] = value;
    }
}