#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Correlation_backward_input1(int item, float *gradInput1, int nInputChannels, int inputHeight, int inputWidth, float *gradOutput, int nOutputChannels, int outputHeight, int outputWidth, float *rInput2, int pad_size, int kernel_size, int max_displacement, int stride1, int stride2) {
    // Calculate coordinates with padding offset
    int n = item;
    int y = blockIdx.x * stride1 + pad_size;
    int x = blockIdx.y * stride1 + pad_size;
    int c = blockIdx.z;
    int tch_off = threadIdx.x;

    // Pre-compute constants
    const int kernel_rad = (kernel_size - 1) / 2;
    const int displacement_rad = max_displacement / stride2;
    const int displacement_size = 2 * displacement_rad + 1;

    int xmin = max(0, (x - kernel_rad - max_displacement) / stride1);
    int ymin = max(0, (y - kernel_rad - max_displacement) / stride1);
    int xmax = min(outputWidth - 1, (x + kernel_rad - max_displacement) / stride1);
    int ymax = min(outputHeight - 1, (y + kernel_rad - max_displacement) / stride1);

    if (xmax < 0 || ymax < 0 || xmin >= outputWidth || ymin >= outputHeight || xmin > xmax || ymin > ymax) return;

    const int pInputWidth = inputWidth + 2 * pad_size;
    const int pdimyxc = pInputWidth * (inputHeight + 2 * pad_size) * nInputChannels;
    const int pdimxc = pInputWidth * nInputChannels;
    const int tdimcyx = nOutputChannels * outputHeight * outputWidth;
    const int tdimyx = outputHeight * outputWidth;
    const int tdimx = outputWidth;
    const int odimcyx = nInputChannels * inputHeight * inputWidth;
    const int odimx = inputWidth;

    float nelems = kernel_size * kernel_size * nInputChannels;

    __shared__ float prod_sum[CUDA_NUM_THREADS];
    prod_sum[tch_off] = 0;

    // Loop unrolling and memory coalescing optimization
    #pragma unroll
    for (int tc = tch_off; tc < nOutputChannels; tc += CUDA_NUM_THREADS) {
        int i2 = (tc % displacement_size - displacement_rad) * stride2;
        int j2 = (tc / displacement_size - displacement_rad) * stride2;

        int indx2 = n * pdimyxc + (y + j2) * pdimxc + (x + i2) * nInputChannels + c;
        float val2 = rInput2[indx2];

        for (int j = ymin; j <= ymax; ++j) {
            for (int i = xmin; i <= xmax; ++i) {
                int tindx = n * tdimcyx + tc * tdimyx + j * tdimx + i;
                prod_sum[tch_off] += gradOutput[tindx] * val2;
            }
        }
    }

    // Threadblock reduction
    __syncthreads();
    if (tch_off == 0) {
        float reduce_sum = 0;
        for (int idx = 0; idx < CUDA_NUM_THREADS; ++idx) {
            reduce_sum += prod_sum[idx];
        }
        const int indx1 = n * odimcyx + c * (inputHeight * inputWidth) + (y - pad_size) * odimx + (x - pad_size);
        gradInput1[indx1] = reduce_sum / nelems;
    }
}