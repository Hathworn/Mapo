#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_tensors(double* SR, const double* fields, const double* norms, const int num_modes, const int Nx) {
    unsigned int full_thread_idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int nmp4 = num_modes * num_modes * num_modes * num_modes;

    // Early exit if thread index is out of bounds
    if (full_thread_idx >= nmp4) {
        return;
    }

    // Pre-calculate these values outside the loop to reduce redundant computations
    unsigned int Nxnm = Nx * num_modes;
    unsigned int midx1 = full_thread_idx % num_modes;
    unsigned int midx2 = (full_thread_idx / num_modes) % num_modes;
    unsigned int midx3 = (full_thread_idx / (num_modes * num_modes)) % num_modes;
    unsigned int midx4 = (full_thread_idx / (num_modes * num_modes * num_modes)) % num_modes;
    
    double sum = 0.0;

    // Unroll loops for potential better performance
    for (int i = 0; i < Nx; i++) {
        unsigned int iimode = i * num_modes;
        for (int j = 0; j < Nx; j++) {
            unsigned int index = iimode + j * Nxnm;
            double f1 = fields[midx1 + index];
            double f2 = fields[midx2 + index];
            double f3 = fields[midx3 + index];
            double f4 = fields[midx4 + index];
            sum += f1 * f2 * f3 * f4;
        }
    }

    // Normalize and store the result
    SR[full_thread_idx] = sum / (norms[midx1] * norms[midx2] * norms[midx3] * norms[midx4]);
}