#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "calculate_sumterm_part.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float2 *Up = NULL;
hipMalloc(&Up, XSIZE*YSIZE);
float2 *Vpl = NULL;
hipMalloc(&Vpl, XSIZE*YSIZE);
const float2 *A_t = NULL;
hipMalloc(&A_t, XSIZE*YSIZE);
const float *SR = NULL;
hipMalloc(&SR, XSIZE*YSIZE);
const unsigned char *nonzero_midx1234s = NULL;
hipMalloc(&nonzero_midx1234s, XSIZE*YSIZE);
const unsigned int N = 1;
const unsigned int M = 1;
const float SK_factor = 1;
const unsigned int NUM_NONZERO = 1;
const unsigned int NUM_MODES = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
calculate_sumterm_part<<<gridBlock, threadBlock>>>(Up,Vpl,A_t,SR,nonzero_midx1234s,N,M,SK_factor,NUM_NONZERO,NUM_MODES);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
calculate_sumterm_part<<<gridBlock, threadBlock>>>(Up,Vpl,A_t,SR,nonzero_midx1234s,N,M,SK_factor,NUM_NONZERO,NUM_MODES);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
calculate_sumterm_part<<<gridBlock, threadBlock>>>(Up,Vpl,A_t,SR,nonzero_midx1234s,N,M,SK_factor,NUM_NONZERO,NUM_MODES);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}