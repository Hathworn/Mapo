#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_sumterm_part(float2 * Up, float2 * Vpl, const float2 * A_t, const float* SR, const unsigned char* nonzero_midx1234s, const unsigned int N, const unsigned int M, const float SK_factor, const unsigned int NUM_NONZERO, const unsigned int NUM_MODES) {
    unsigned int full_thread_idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Calculate the index
    unsigned int Midx = full_thread_idx / N;
    unsigned int Nidx = full_thread_idx % N;
    unsigned int NM = N * M;

    if (full_thread_idx >= NM) {
        return;
    }

    // Pre-calculate the base index to reduce redundant calculations
    unsigned int baseIndex = Nidx + Midx * N;

    // Compute the tensors
    for (int i = 0; i < NUM_NONZERO; i++) {
        unsigned int midx1 = nonzero_midx1234s[i * 4] - 1;
        unsigned int midx2 = nonzero_midx1234s[1 + i * 4] - 1;
        unsigned int midx3 = nonzero_midx1234s[2 + i * 4] - 1;
        unsigned int midx4 = nonzero_midx1234s[3 + i * 4] - 1;

        // Pre-compute indices to reduce repetitive calculations
        unsigned int idx2 = baseIndex + midx2 * NM;
        unsigned int idx3 = baseIndex + midx3 * NM;
        unsigned int idx4 = baseIndex + midx4 * NM;
        unsigned int updateIndex = baseIndex + midx1 * NM;
        unsigned int vplIndex = updateIndex + midx2 * NM * NUM_MODES;

        float a = A_t[idx2].x;
        float b = A_t[idx2].y;
        float c = A_t[idx3].x;
        float d = A_t[idx3].y;
        float e = A_t[idx4].x;
        float f = A_t[idx4].y;

        // Use temporary variables to reduce the number of accesses to Up and Vpl
        float upX = Up[updateIndex].x + SK_factor * SR[i] * (a * c * e - b * d * e + a * d * f + c * b * f);
        float upY = Up[updateIndex].y + SK_factor * SR[i] * (a * d * e + c * b * e - a * c * f + b * d * f);
        float vplX = Vpl[vplIndex].x + SR[i] * (c * e + d * f);
        float vplY = Vpl[vplIndex].y + SR[i] * (d * e - c * f);

        // Assign the previously cached values back to Up and Vpl
        Up[updateIndex].x = upX;
        Up[updateIndex].y = upY;
        Vpl[vplIndex].x = vplX;
        Vpl[vplIndex].y = vplY;
    }
}