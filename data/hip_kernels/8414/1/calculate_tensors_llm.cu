#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_tensors(float* SR, const float* fields, const float* norms, const int num_modes, const int Nx) {
    unsigned int full_thread_idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Calculate the index
    unsigned int nmp4 = num_modes * num_modes * num_modes * num_modes;
    unsigned int Nxnm = Nx * num_modes;

    if (full_thread_idx >= nmp4) {
        return;
    }

    // Turn linear index into components
    unsigned int midx1 = full_thread_idx % num_modes;
    unsigned int midx2 = (full_thread_idx / num_modes) % num_modes;
    unsigned int midx3 = (full_thread_idx / num_modes / num_modes) % num_modes;
    unsigned int midx4 = (full_thread_idx / num_modes / num_modes / num_modes);

    float sum = 0.0f;
    // Use registers for accumulation
    for (int i = 0; i < Nx; i++) {
        for (int j = 0; j < Nx; j++) {
            // Load field values once per iteration
            float field1 = fields[midx1 + i * num_modes + j * Nxnm];
            float field2 = fields[midx2 + i * num_modes + j * Nxnm];
            float field3 = fields[midx3 + i * num_modes + j * Nxnm];
            float field4 = fields[midx4 + i * num_modes + j * Nxnm];
            sum += field1 * field2 * field3 * field4;
        }
    }

    // Normalize
    float norm = norms[midx1] * norms[midx2] * norms[midx3] * norms[midx4];
    SR[full_thread_idx] = sum / norm; // Store normalized result
}