#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_sumterm_part(double2* Up, double2* Vpl, const double2* A_t, const double* SR, const unsigned char* nonzero_midx1234s, const unsigned int N, const unsigned int M, const double SK_factor, const unsigned int NUM_NONZERO, const unsigned int NUM_MODES) {
    unsigned int full_thread_idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int NM = N * M;

    // Early exit if out of bounds
    if (full_thread_idx >= NM) {
        return;
    }

    // Calculate index
    unsigned int Midx = full_thread_idx / N;
    unsigned int Nidx = full_thread_idx % N;

    // Compute the tensors
    for (int i = 0; i < NUM_NONZERO; ++i) {
        unsigned int midx1 = nonzero_midx1234s[i * 4] - 1;
        unsigned int midx2 = nonzero_midx1234s[1 + i * 4] - 1;
        unsigned int midx3 = nonzero_midx1234s[2 + i * 4] - 1;
        unsigned int midx4 = nonzero_midx1234s[3 + i * 4] - 1;

        unsigned int idx2 = Nidx + Midx * N + midx2 * NM;
        unsigned int idx3 = Nidx + Midx * N + midx3 * NM;
        unsigned int idx4 = Nidx + Midx * N + midx4 * NM;

        double a = A_t[idx2].x;
        double b = A_t[idx2].y;
        double c = A_t[idx3].x;
        double d = A_t[idx3].y;
        double e = A_t[idx4].x;
        double f = A_t[idx4].y;

        unsigned int idx1 = Nidx + Midx * N + midx1 * NM;
        double2 tempUp = Up[idx1];
        
        // Update Up
        tempUp.x += SK_factor * SR[i] * (a * c * e - b * d * e + a * d * f + c * b * f);
        tempUp.y += SK_factor * SR[i] * (a * d * e + c * b * e - a * c * f + b * d * f);
        Up[idx1] = tempUp;

        // Update Vpl
        unsigned int idxVpl = idx1 + midx2 * NM * NUM_MODES;
        double2 tempVpl = Vpl[idxVpl];
        tempVpl.x += SR[i] * (c * e + d * f);
        tempVpl.y += SR[i] * (d * e - c * f);
        Vpl[idxVpl] = tempVpl;
    }
}