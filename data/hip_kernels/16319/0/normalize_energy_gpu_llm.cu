#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_energy_gpu(float *ksn2e, float *ksn2f, double omega_re, double omega_im, float *nm2v_re, float *nm2v_im, int nfermi, int norbs, int nvirt, int vstart)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // nocc
    int j = blockIdx.y * blockDim.y + threadIdx.y; // nvirt

    // Early exit if indices are out of bounds for improved efficiency
    if (i >= nfermi || j >= norbs - vstart) return;

    float en = ksn2e[i];
    float fn = ksn2f[i];
    float em = ksn2e[j + vstart];
    float fm = ksn2f[j + vstart];

    double d1p = omega_re - (em - en);
    double d1pp = omega_im;
    double d2p = omega_re + (em - en);
    double d2pp = omega_im;

    double d1p_denom = d1p*d1p + d1pp*d1pp;
    double d2p_denom = d2p*d2p + d2pp*d2pp;

    // Precompute factors to minimize arithmetic operations
    double alpha = d1p/d1p_denom - d2p/d2p_denom;
    double beta = -d1pp/d1p_denom + d2pp/d2p_denom;

    float old_re = nm2v_re[i*nvirt + j];
    float old_im = nm2v_im[i*nvirt + j];

    // Update nm2v_re and nm2v_im using precomputed factors
    nm2v_re[i*nvirt + j] = (fn - fm) * (old_re * alpha - old_im * beta);
    nm2v_im[i*nvirt + j] = (fn - fm) * (old_re * beta + old_im * alpha);
}