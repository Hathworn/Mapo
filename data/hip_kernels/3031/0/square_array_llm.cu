#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square_array(float *a, int N)
{
    // Calculate the unique index for this thread
    int idx = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;

    // Use a while loop to handle larger problem sizes beyond the grid dimensions
    while (idx < N) {
        a[idx] = a[idx] * a[idx]; // Square the element
        idx += hipBlockDim_x * hipGridDim_x; // Move to the next index for this thread
    }
}