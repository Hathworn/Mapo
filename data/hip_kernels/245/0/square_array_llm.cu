#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(double *a, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        // Efficiently square the element
        double value = a[idx];
        a[idx] = value * value;
        printf("idx = %d, a = %f\n", idx, a[idx]);  // Debug output, can be removed for performance
    }
}