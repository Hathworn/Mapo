#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void double_value(double *x, double *y)
{
    // Use blockIdx and blockDim for scalable indexing
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    y[idx] = 2.0 * x[idx];
}
```
