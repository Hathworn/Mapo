#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refine_dilateFPPlaneDepthMapXpYp_kernel(float* fpPlaneDepthMap, int fpPlaneDepthMap_p, float* maskMap, int maskMap_p, int width, int height, int xp, int yp, float fpPlaneDepth)
{
    // Calculate the global thread coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Avoid redundant boundary checks and ensure coordinates are within bounds
    if((x < width) && (y < height))
    {
        // Reuse calculated depth value if condition is met
        float depth = maskMap[y * maskMap_p + x];
        if(depth > 0.0f)
        {
            int new_x = x + xp;
            int new_y = y + yp;
            
            // Final check to ensure the modified coordinates are within bounds
            if((new_x >= 0) && (new_y >= 0) && (new_x < width) && (new_y < height))
            {
                fpPlaneDepthMap[new_y * fpPlaneDepthMap_p + new_x] = fpPlaneDepth;
            }
        }
    }
}