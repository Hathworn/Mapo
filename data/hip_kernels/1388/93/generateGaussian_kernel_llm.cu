#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generateGaussian_kernel(float* og, float delta, int radius)
{
    int idx = threadIdx.x; // Cache threadIdx.x
    int x = idx - radius;
    float delta2 = 2 * delta * delta; // Precompute 2 * delta^2
    og[idx] = __expf(-(x * x) / delta2); // Use precomputed delta2
}