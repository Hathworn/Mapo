#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refine_fuseThreeDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float* isimLst, int isimLst_p, float* idptLst, int idptLst_p, float* isimAct, int isimAct_p, float* idptAct, int idptAct_p, int width, int height, float simThr)
{
    // Calculate coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if((x < width) && (y < height))
    {
        // Utilize shared memory for faster access
        __shared__ float sharedDpts[3];
        __shared__ float sharedSims[3];

        sharedDpts[0] = odpt[y * odpt_p + x];
        sharedDpts[1] = idptLst[y * idptLst_p + x];
        sharedDpts[2] = idptAct[y * idptAct_p + x];

        sharedSims[0] = osim[y * osim_p + x];
        sharedSims[1] = isimLst[y * isimLst_p + x];
        sharedSims[2] = isimAct[y * isimAct_p + x];

        float outDpt = sharedDpts[0];
        float outSim = simThr;

        // Simplified conditional assignments using ternary operations
        outDpt = ((sharedSims[1] < outSim) ? sharedDpts[1] : outDpt);
        outSim = ((sharedSims[1] < outSim) ? sharedSims[1] : outSim);
        outDpt = ((sharedSims[2] < outSim) ? sharedDpts[2] : outDpt);
        outSim = ((sharedSims[2] < outSim) ? sharedSims[2] : outSim);

        // Write back results
        osim[y * osim_p + x] = outSim;
        odpt[y * odpt_p + x] = outDpt;
    }
}