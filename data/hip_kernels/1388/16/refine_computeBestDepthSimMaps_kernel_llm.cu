#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float refineDepthSubPixel(const float3& depths, const float3& sims)
{
    float outDepth = -1.0f;

    // Normalize similarities
    float simM1 = (sims.x + 1.0f) / 2.0f;
    float simP1 = (sims.z + 1.0f) / 2.0f;
    float sim1 = (sims.y + 1.0f) / 2.0f;

    // Check condition for subpixel refinement
    if((simM1 > sim1) && (simP1 > sim1))
    {
        // Calculate disparity step using quadratic interpolation
        float dispStep = -((simP1 - simM1) / (2.0f * (simP1 + simM1 - 2.0f * sim1)));

        float floatDepthM1 = depths.x;
        float floatDepthP1 = depths.z;

        // Calculate linear function coefficients
        float b = (floatDepthP1 + floatDepthM1) / 2.0f;
        float a = b - floatDepthM1;

        // Apply linear function to refine depth
        outDepth = a * dispStep + b;
    }

    return outDepth;
}

__global__ void refine_computeBestDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float3* isims, int isims_p, float3* idpts, int idpts_p, int width, int height, float simThr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < width) && (y < height))
    {
        float3 depths = idpts[y * idpts_p + x];
        float3 sims = isims[y * isims_p + x];

        // Determine initial best depth and similarity
        float outDepth = ((sims.x < sims.y) ? depths.x : depths.y);
        float outSim = ((sims.x < sims.y) ? sims.x : sims.y);
        outDepth = ((sims.z < outSim) ? depths.z : outDepth);
        outSim = ((sims.z < outSim) ? sims.z : outSim);

        // Refine depth if possible
        float refinedDepth = refineDepthSubPixel(depths, sims);
        if (refinedDepth > 0.0f)
        {
            outDepth = refinedDepth;
        }

        // Update output maps based on threshold
        osim[y * osim_p + x] = (outSim < simThr ? outSim : 1.0f);
        odpt[y * odpt_p + x] = (outSim < simThr ? outDepth : -1.0f);
    }
}