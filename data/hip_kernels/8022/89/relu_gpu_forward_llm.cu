#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Use shared memory for inputs
    extern __shared__ float shared_data[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    if (tid < N) {
        // Load data into shared memory
        shared_data[local_tid] = in[tid];
        __syncthreads();

        // Perform ReLU
        out[tid] = shared_data[local_tid] > 0 ? shared_data[local_tid] : 0;
    }
}