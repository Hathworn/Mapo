#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Process the elements using multiple threads in parallel
    for (int idx = tid; idx < N; idx += blockDim.x * gridDim.x) {
        ingrad[idx] = indata[idx] > 0 ? outgrad[idx] : 0;
    }
}