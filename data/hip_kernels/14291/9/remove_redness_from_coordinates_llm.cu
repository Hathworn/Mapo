#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void remove_redness_from_coordinates(const unsigned int* d_coordinates, unsigned char* d_r, unsigned char* d_b, unsigned char* d_g, unsigned char* d_r_output, int num_coordinates, int num_pixels_y, int num_pixels_x, int template_half_height, int template_half_width) {
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;
    int imgSize = num_pixels_x * num_pixels_y;

    if (global_index_1d < num_coordinates) {
        unsigned int image_index_1d = d_coordinates[imgSize - global_index_1d - 1];
        ushort2 image_index_2d = make_ushort2(image_index_1d % num_pixels_x, image_index_1d / num_pixels_x);

        // Pre-calculate clamp bounds
        int y_start = max(0, image_index_2d.y - template_half_height);
        int y_end = min(num_pixels_y - 1, image_index_2d.y + template_half_height);
        int x_start = max(0, image_index_2d.x - template_half_width);
        int x_end = min(num_pixels_x - 1, image_index_2d.x + template_half_width);

        for (int y = y_start; y <= y_end; y++) {
            for (int x = x_start; x <= x_end; x++) {
                int image_offset_index_1d_clamped = y * num_pixels_x + x;

                // Combine loading and averaging
                d_r_output[image_offset_index_1d_clamped] = (d_g[image_offset_index_1d_clamped] + d_b[image_offset_index_1d_clamped]) / 2;
            }
        }
    }
}