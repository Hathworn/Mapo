#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shmem_reduce_kernel(float *d_out, const float *d_in) {
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // Ensure myId is within bounds before loading
    sdata[tid] = (myId < gridDim.x * blockDim.x) ? d_in[myId] : 0.0f;
    __syncthreads();

    // Use unrolling for the last warp if needed
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Unrolling warp loop for efficiency
    if (tid < 32) {
        volatile float *vsmem = sdata; // Volatile to prevent reordering
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}