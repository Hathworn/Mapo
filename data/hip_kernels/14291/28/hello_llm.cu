#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Utilize shared memory to minimize global memory access
    __shared__ int blockId;
    // Load block index into shared memory
    if (threadIdx.x == 0) {
        blockId = blockIdx.x;
    }
    __syncthreads();
    // Print using shared memory variable
    printf("Hello world! I'm a thread in block %d\n", blockId);
}