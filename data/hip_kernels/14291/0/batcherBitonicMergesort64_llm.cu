#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    // Perform bitonic sort
    for (int stage = 0; stage <= 5; stage++)
    {
        for (int substage = stage; substage >= 0; substage--)
        {
            int pos = 1 << substage; // Calculate position offset
            int j = (tid >> substage) << (substage + 1); // Calculate pair index
            int direction = (((tid >> stage) & 2) == 0); // Determine direction

            // Compare and swap if needed based on direction
            if ((j + pos) < 64) // Ensure within bounds
            {
                if ((sdata[j] > sdata[j + pos]) == direction)
                {
                    float temp = sdata[j];
                    sdata[j] = sdata[j + pos];
                    sdata[j + pos] = temp;
                }
            }
            __syncthreads();
        }
    }

    d_out[tid] = sdata[tid]; // Store sorted data to output
}