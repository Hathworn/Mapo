#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized Hello World kernel
__global__ void hello()
{
    // Calculate unique global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use the global thread index in printf for better clarity
    printf("Hello world! I'm thread %d in block %d\n", idx, blockIdx.x);
}