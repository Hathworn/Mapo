#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tonemap(float* d_x, float* d_y, float* d_log_Y, float* d_cdf_norm, float* d_r_new, float* d_g_new, float* d_b_new, float min_log_Y, float max_log_Y, float log_Y_range, int num_bins, int num_pixels_y, int num_pixels_x)
{
    int ny = num_pixels_y;
    int nx = num_pixels_x;
    int2 image_index_2d = make_int2((blockIdx.x * blockDim.x) + threadIdx.x, (blockIdx.y * blockDim.y) + threadIdx.y);

    // Ensure valid pixel range using a single comparison
    if (image_index_2d.x >= nx || image_index_2d.y >= ny) return;

    int image_index_1d = (nx * image_index_2d.y) + image_index_2d.x;
    
    float x = d_x[image_index_1d];
    float y = d_y[image_index_1d];
    float log_Y = d_log_Y[image_index_1d];
    int bin_index = min(num_bins - 1, int((num_bins * (log_Y - min_log_Y)) / log_Y_range));
    float Y_new = d_cdf_norm[bin_index];

    // Precompute reciprocal of y for reuse
    float y_inv = 1.0f / y;
    float X_new = x * (Y_new * y_inv);
    float Z_new = (1 - x - y) * (Y_new * y_inv);

    d_r_new[image_index_1d] = (X_new * 3.2406f) + (Y_new * -1.5372f) + (Z_new * -0.4986f);
    d_g_new[image_index_1d] = (X_new * -0.9689f) + (Y_new * 1.8758f) + (Z_new * 0.0415f);
    d_b_new[image_index_1d] = (X_new * 0.0557f) + (Y_new * -0.2040f) + (Z_new * 1.0570f);
}