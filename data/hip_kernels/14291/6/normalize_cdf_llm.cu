#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
{
    const float normalization_constant = 1.f / d_input_cdf[n - 1];
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimization: Ensure all threads have a valid index
    if (global_index_1d < n)
    {
        // Optimization: Reduce register usage by using fewer local variables
        d_output_cdf[global_index_1d] = d_input_cdf[global_index_1d] * normalization_constant;
    }
}