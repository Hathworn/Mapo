#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Calculate global thread index
    int myId = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Fetch data item
    int myItem = d_in[myId];

    // Compute bin index
    int myBin = myItem % BIN_COUNT;

    // Use atomic operation to increment bin count safely across threads
    atomicAdd(&d_bins[myBin], 1);
}