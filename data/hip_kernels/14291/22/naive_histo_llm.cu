#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naive_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Use shared memory for atomic operation buffering
    extern __shared__ int shared_bins[];
    
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;

    // Initialize shared memory
    if (threadIdx.x < BIN_COUNT) {
        shared_bins[threadIdx.x] = 0;
    }
    __syncthreads();

    // Use atomicAdd within shared memory
    atomicAdd(&shared_bins[myBin], 1);
    __syncthreads();

    // Transfer results from shared memory to global memory
    if (threadIdx.x < BIN_COUNT) {
        atomicAdd(&d_bins[threadIdx.x], shared_bins[threadIdx.x]);
    }
}