#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
    extern __shared__ float shared_data[]; // Use shared memory
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // Load data into shared memory
    shared_data[tid] = d_in[myId];
    __syncthreads(); // Ensure all data is loaded

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads(); // Ensure all adds at one stage are done
    }

    // Only thread 0 writes result for this block back to global memory
    if (tid == 0)
    {
        d_out[blockIdx.x] = shared_data[0];
    }
}