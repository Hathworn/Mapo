#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (index >= N) return;

    float denom = 1.f - powf(B1, t); // Precompute denominator
    float sqrt_b2t = sqrtf(1.f - powf(B2, t)); // Precompute sqrt of 1 - B2^t
    float sqrt_v_eps = sqrtf(v[index]) + eps; // Precompute sqrt(v)+eps

    x[index] += (rate * sqrt_b2t / denom) * m[index] / sqrt_v_eps; // Simplified x[index] update
}