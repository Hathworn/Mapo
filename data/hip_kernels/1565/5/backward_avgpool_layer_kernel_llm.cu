#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // 1D indexing
    if (id >= n * w * h) return;  // Return if out of bounds

    int wh = w * h;
    int offset = id / wh;  // Calculate offset for channels and batch
    int i = id % wh;  // Calculate local index within w*h

    int k = offset % c;
    int b = offset / c;

    int in_index = i + wh * (k + b * c);  // Flattened index
    int out_index = k + c * b;

    in_delta[in_index] += out_delta[out_index] / wh;  // Perform operation
}