#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) return;

    // Use shared memory to reduce global memory accesses
    extern __shared__ float shared_sum[];
    shared_sum[threadIdx.x] = 0.0;

    // Accumulate in shared memory
    for (int b = 0; b < batch; ++b) {
        int i = b * n + index;
        shared_sum[threadIdx.x] += delta[i];
    }

    __syncthreads(); // Ensure all threads complete shared memory accumulation
    
    // Reduce within block
    if (threadIdx.x == 0) {
        float blockSum = 0.0;
        for (int i = 0; i < blockDim.x; ++i) {
            blockSum += shared_sum[i];
        }
        atomicAdd(&bias_updates[blockIdx.x], blockSum);
    }
}