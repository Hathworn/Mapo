#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0;

    // Unrolling the outer loop for better performance
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {  // Distribute iterations directly among threads
            int index = j * spatial * filters + filter * spatial + i;
            float diff = x[index] - mean[filter];
            local[id] += diff * diff;
        }
    }

    __syncthreads();

    // Reduction: Summing up values in shared memory
    if (id == 0) {
        float sum = 0;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        variance[filter] = sum / (spatial * batch - 1);
    }
}