#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation for 1D grid
    if (index >= N) return;

    int b = index / (layers * spatial);
    int in_c = (index / spatial) % layers;
    int in_s = index % spatial;

    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;

    if (forward) {
        out[i2] = x[i1]; // Optimized memory write for forward direction
    } else {
        out[i1] = x[i2]; // Optimized memory write for backward direction
    }
}