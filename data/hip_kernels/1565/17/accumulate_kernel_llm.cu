#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    // Initialize shared memory for block-level parallel reduction
    extern __shared__ float shared_sum[];
    shared_sum[threadIdx.x] = 0;

    // Accumulate with a stride to improve coalesced access and utilize shared mem
    for (int k = threadIdx.x; k < n; k += blockDim.x) {
        shared_sum[threadIdx.x] += x[k * groups + i];
    }

    // Synchronize to ensure all threads have completed their local sums
    __syncthreads();

    // Perform block-level reduction to sum into the global sum array
    if (threadIdx.x == 0) {
        for (int k = 1; k < blockDim.x; ++k) {
            shared_sum[0] += shared_sum[k];
        }
        sum[i] = shared_sum[0];
    }
}