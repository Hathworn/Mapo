#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Ensure thread processes only valid elements
    if(i < N) {
        // Load value into a register
        float val = X[i * INCX];

        // Perform computation and write-back conditionally
        if(val * val < ALPHA * ALPHA) {
            X[i * INCX] = 0;
        }
    }
}