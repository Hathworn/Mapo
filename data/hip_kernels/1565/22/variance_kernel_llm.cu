#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1.f / (batch * spatial - 1);
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified block index calculation
    if (i >= filters) return;
    
    float sum = 0.0f; // Use a local variable to accumulate the variance
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += powf((x[index] - mean[i]), 2); // Accumulate variance in the local variable
        }
    }
    variance[i] = sum * scale; // Assign the scaled result back to global memory
}