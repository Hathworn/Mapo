#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Check index within bounds and modify X
    if(i < N) 
    {
        float val = X[i * INCX];
        X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, val));
    }
}