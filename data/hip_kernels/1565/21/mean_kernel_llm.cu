#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    extern __shared__ float shared_mem[]; // Shared memory for reduction
    float scale = 1.f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float mean_val = 0.0f; // Initialize mean value

    // Reduce batch and spatial dimensions
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            mean_val += x[index];
        }
    }

    // Store partial result in shared memory
    shared_mem[threadIdx.x] = mean_val;
    __syncthreads();

    // Reduce within the block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            shared_mem[threadIdx.x] += shared_mem[threadIdx.x + s];
        }
        __syncthreads();
    }
    
    // Store the result for this filter index
    if (threadIdx.x == 0) {
        mean[blockIdx.x] = shared_mem[0] * scale;
    }
}