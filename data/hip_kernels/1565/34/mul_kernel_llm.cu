#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x; 

    // Use grid-stride loop for handling large data and maximizing occupancy
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) 
    {
        Y[i * INCY] *= X[i * INCX];
    }
}