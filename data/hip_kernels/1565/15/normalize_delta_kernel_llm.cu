#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if index is out of bounds
    if (index >= N) return;

    // Calculate the filter index
    int f = (index / spatial) % filters;

    // Pre-compute values used in the calculation to reduce redundant computations
    float inv_sqrt_var = rsqrtf(variance[f] + .00001f);
    float scale_factor = 2.0f / (spatial * batch);

    // Update delta with optimized expression
    delta[index] = delta[index] * inv_sqrt_var + variance_delta[f] * scale_factor * (x[index] - mean[f]) + mean_delta[f] / (spatial * batch);
}