#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) return;
    
    // Precompute commonly used multiplications
    int sample_w2 = sample * w2;
    int stride_w1 = stride * w1;

    int i = id % minw;
    id /= minw;
    int j = id % minh;
    id /= minh;
    int k = id % minc;
    id /= minc;
    int b = id % batch;

    // Improved index calculation with precomputed values
    int out_index = i * sample + sample_w2 * j + h2 * (k + c2 * b);
    int add_index = i * stride + stride_w1 * j + h1 * (k + c1 * b);
    
    // Perform the addition
    atomicAdd(&out[out_index], add[add_index]);  // Use atomic addition to prevent race condition
}