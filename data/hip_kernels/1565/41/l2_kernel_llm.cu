#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate index without blockIdx.y

    // Use a loop to process all relevant indices
    for (; i < n; i += blockDim.x * gridDim.x) {
        float diff = truth[i] - pred[i];
        error[i] = diff * diff;  // Calculate squared difference for error
        delta[i] = diff;         // Calculate difference for delta
    }
}