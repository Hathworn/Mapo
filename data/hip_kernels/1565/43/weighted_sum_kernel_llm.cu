#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Use shared memory to reduce global memory access
    extern __shared__ float shared_s[];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = gridDim.x * blockDim.x;

    while (i < n) {
        shared_s[threadIdx.x] = s[i];
        __syncthreads(); // Synchronize shared memory between threads

        // Minimize conditional execution within the loop
        c[i] = shared_s[threadIdx.x] * a[i] + (1 - shared_s[threadIdx.x]) * (b ? b[i] : 0);

        i += gridStride; // Advance to the next set of data within the grid stride
    }
}