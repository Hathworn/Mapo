#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation
    int totalSize = (NX + NY) * B; // Precompute total size
    if(i < totalSize){
        int b = i / (NX+NY); // Compute batch index
        int j = i % (NX+NY); // Compute index within batch
        OUT[i] = (j < NX) ? X[b*NX + j] : Y[b*NY + j - NX]; // Use ternary operator for conditional
    }
}