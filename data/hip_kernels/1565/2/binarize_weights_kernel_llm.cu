#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Use shared memory to compute the mean more efficiently
    extern __shared__ float shared_weights[];

    float sum = 0.0f;
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[i] = fabsf(weights[f * size + i]);
        sum += shared_weights[i];
    }

    // Reduce sum within the block
    __shared__ float block_sum;
    block_sum = 0.0f;
    __syncthreads();

    atomicAdd(&block_sum, sum);
    __syncthreads();

    float mean = block_sum / size;

    // Assign binary values
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    }
}