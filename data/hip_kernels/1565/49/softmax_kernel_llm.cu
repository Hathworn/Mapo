#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0.0f;
    float largest = -INFINITY;
    
    // Use threads to find the largest value
    for (i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = fmaxf(largest, val);
    }
    
    for (i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp);
        sum += e;
        output[i * stride] = e;
    }
    
    float inv_sum = 1.0f / sum;
    
    // Normalize using precomputed inverse of sum for efficiency
    for (i = 0; i < n; ++i) {
        output[i * stride] *= inv_sum;
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= batch * groups) return;
    
    int b = id / groups;
    int g = id % groups;
    
    // Use cooperative loading of batch and group data
    softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}