#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Use shared memory for faster access
    extern __shared__ float shared_x[];
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    if (i < N) {
        // Load data into shared memory
        shared_x[threadIdx.x] = X[OFFX + i * INCX];
        __syncthreads();  // Ensure all threads have loaded their data

        // Perform the operation using shared memory
        Y[OFFY + i * INCY] += ALPHA * shared_x[threadIdx.x];
    }
}