#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Efficient index calculation
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        // Memory coalescing by ensuring contiguous memory access
        X[i * INCX] = ALPHA;
    }
}