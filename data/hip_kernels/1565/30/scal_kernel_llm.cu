#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use flat 1D block index calculation for simplicity
    int stride = blockDim.x * gridDim.x; // Calculate stride for handling larger arrays
    for(int idx = i; idx < N; idx += stride) // Use loop for processing multiple elements
    {
        X[idx * INCX] *= ALPHA; // Multiply current element by ALPHA
    }
}