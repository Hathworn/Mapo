#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    // Use shared memory for partial sum accumulation.
    extern __shared__ float shared_delta[];
    
    int tx = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tx;
    shared_delta[tx] = 0;
    
    // Reduce unnecessary calculations by using the step size of block size.
    for (int j = 0; j < batch; ++j) {
        for (int k = tx; k < spatial; k += blockDim.x) {
            int global_index = j * filters * spatial + blockIdx.x * spatial + k;
            if (global_index < batch * filters * spatial) {
                shared_delta[tx] += delta[global_index];
            }
        }
    }
    
    __syncthreads();

    // Reduction within block
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tx < stride) {
            shared_delta[tx] += shared_delta[tx + stride];
        }
        __syncthreads();
    }
    
    // Only thread 0 writes result back to global memory
    if (tx == 0) {
        mean_delta[blockIdx.x] = shared_delta[0] * (-1.f / sqrtf(variance[blockIdx.x] + .00001f));
    }
}