#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCK 1024

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Unroll loop for improved performance
    for(int b = 0; b < batch; ++b){
        for(int i = 0; i < size; i += BLOCK){
            int index = p + i + size * (filter + n * b);
            if ((p + i) < size) {
                sum += delta[index] * x_norm[index];
            }
        }
    }
    part[p] = sum;
    __syncthreads();

    // Use atomicAdd for thread-safe accumulation
    if (p == 0) {
        for(int i = 0; i < BLOCK; ++i) {
            atomicAdd(&scale_updates[filter], part[i]);
        }
    }
}