#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop through all elements in a grid-strided manner
    for (int idx = index; idx < n * size * batch; idx += stride) {
        int i = idx % size;
        int j = (idx / size) % n;
        int k = idx / (size * n);

        output[(k * n + j) * size + i] += biases[j];
    }
}