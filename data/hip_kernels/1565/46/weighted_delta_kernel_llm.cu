#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Utilize shared memory for frequently accessed data
    extern __shared__ float shared_s[];

    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    // Load shared memory only if index is within bounds
    if (i < n) shared_s[threadIdx.x] = s[i];

    // Ensure cache consistency
    __syncthreads();

    if (i < n) {
        float Si = shared_s[threadIdx.x];
        float Dci = dc[i];

        if (da) da[i] += Dci * Si;
        if (db) db[i] += Dci * (1 - Si);
        ds[i] += Dci * (a[i] - b[i]);
    }
}