#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel with optimized looping and memory access patterns
__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    // Calculate 2D indexes and thread ID based on block/thread hierarchy
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) return;

    // Calculate 3D indexes based on flat thread/block IDs
    int w_offset = -pad;
    int h_offset = -pad;

    int j = index % in_w;
    int i = (index / in_w) % in_h;
    int k = (index / (in_w * in_h)) % in_c;
    int b = index / (in_w * in_h * in_c);

    // Initialize delta sum for the current index
    float d = 0.0f;

    // Calculate output dimensions
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int c = in_c;

    // Area for each pooling operation
    int area = (size - 1) / stride;

    // Loop through window area using register space for local indices
    for (int l = -area; l <= area; ++l) {
        for (int m = -area; m <= area; ++m) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;

            // Calculate flat index for output (pool-grid dimensions)
            int out_index = out_w + w * (out_h + h * (k + c * b));

            // Check if outputs are within valid grid range and accumulate valid deltas
            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h) {
                if (indexes[out_index] == index) {
                    d += delta[out_index];
                }
            }
        }
    }

    // Update prev_delta with accumulated delta
    prev_delta[index] += d;
}