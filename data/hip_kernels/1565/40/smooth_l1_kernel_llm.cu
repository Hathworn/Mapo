#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Grid stride loop for improved memory coalescing and load balance
__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);
        if(abs_val < 1) {
            error[i] = diff * diff;
            delta[i] = diff;
        } else {
            error[i] = 2*abs_val - 1;
            delta[i] = (diff > 0) ? 1 : -1;
        }
    }
}