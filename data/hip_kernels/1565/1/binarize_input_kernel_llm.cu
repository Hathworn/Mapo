#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    // Calculate global thread index for 1D grid
    int s = blockIdx.x * blockDim.x + threadIdx.x;
    if (s >= size) return; // Boundary check

    float mean = 0.0f;
    for(int i = 0; i < n; ++i){
        mean += fabsf(input[i * size + s]);
    }
    mean /= n;

    float binary_val = (input[s] > 0) ? mean : -mean; // Calculate binary value based on sign
    for(int i = 0; i < n; ++i){
        binary[i * size + s] = binary_val; // Assign precomputed binary value
    }
}