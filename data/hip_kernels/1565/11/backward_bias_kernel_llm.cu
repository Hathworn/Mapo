#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Use loop unrolling for better performance
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; i += BLOCK) {
            int index = p + i + size * (filter + n * b);
            sum += (p + i < size) ? delta[index] : 0;
        }
    }

    part[p] = sum;
    __syncthreads();

    // Use reduction for optimal summation
    if (p < 32) part[p] += part[p + 32];
    __syncthreads();
    if (p < 16) part[p] += part[p + 16];
    __syncthreads();
    if (p < 8) part[p] += part[p + 8];
    __syncthreads();
    if (p < 4) part[p] += part[p + 4];
    __syncthreads();
    if (p < 2) part[p] += part[p + 2];
    __syncthreads();
    if (p == 0) {
        bias_updates[filter] += part[0] + part[1];
    }
}