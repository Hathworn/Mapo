#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    float h = hsv.x;
    float s = hsv.y;
    float v = hsv.z;

    float r, g, b;
    float f, p, q, t;

    if (s == 0) {
        r = g = b = v;
    } else {
        int index = (int) floorf(h);
        f = h - index;
        p = v * (1 - s);
        q = v * (1 - s * f);
        t = v * (1 - s * (1 - f));
        switch (index) {  // Use switch for clearer control flow
            case 0: r = v; g = t; b = p; break;
            case 1: r = q; g = v; b = p; break;
            case 2: r = p; g = v; b = t; break;
            case 3: r = p; g = q; b = v; break;
            case 4: r = t; g = p; b = v; break;
            default: r = v; g = p; b = q; break;
        }
    }

    r = fminf(fmaxf(r, 0), 1);  // Clamp values using fminf and fmaxf
    g = fminf(fmaxf(g, 0), 1);
    b = fminf(fmaxf(b, 0), 1);
    return make_float3(r, g, b);
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    float r = rgb.x;
    float g = rgb.y;
    float b = rgb.z;

    float h, s, v;
    float max = fmaxf(r, fmaxf(g, b));  // Use fmaxf
    float min = fminf(r, fminf(g, b));  // Use fminf
    float delta = max - min;

    v = max;
    if(max == 0){
        s = 0;
        h = -1;
    } else {
        s = delta / max;
        if(r == max){
            h = (g - b) / delta;
        } else if (g == max) {
            h = 2 + (b - r) / delta;
        } else {
            h = 4 + (r - g) / delta;
        }
        if (h < 0) h += 6;
    }
    return make_float3(h, s, v);
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int size = batch * w * h;
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= size) return;

    int idx_temp = id;  // Capture id initially
    int x = idx_temp % w;
    idx_temp /= w;
    int y = idx_temp % h;
    idx_temp /= h;
    
    float rshift = rand[0];
    float gshift = rand[1];
    float bshift = rand[2];
    float r0 = rand[8*id + 0];
    float r1 = rand[8*id + 1];
    float r2 = rand[8*id + 2];
    float r3 = rand[8*id + 3];

    saturation = r0 * (saturation - 1) + 1;
    saturation = (r1 > 0.5f) ? 1.0f / saturation : saturation;
    exposure = r2 * (exposure - 1) + 1;
    exposure = (r3 > 0.5f) ? 1.0f / exposure : exposure;

    size_t offset = id * h * w * 3;
    image += offset;
    float r = image[x + w * (y + h * 0)];
    float g = image[x + w * (y + h * 1)];
    float b = image[x + w * (y + h * 2)];
    float3 rgb = make_float3(r, g, b);

    if(train){
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }

    float shift_scale = scale + (shift * 0.5f);  // Precompute shift scaling
    image[x + w * (y + h * 0)] = rgb.x * scale + translate + (rshift - 0.5f) * shift_scale;
    image[x + w * (y + h * 1)] = rgb.y * scale + translate + (gshift - 0.5f) * shift_scale;
    image[x + w * (y + h * 2)] = rgb.z * scale + translate + (bshift - 0.5f) * shift_scale;
}