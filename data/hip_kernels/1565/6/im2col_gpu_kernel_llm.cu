#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void im2col_gpu_kernel(const int n, const float* data_im, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_col) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index < n) { // Ensure threads operate within bounds
        // Calculate indices and offsets
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;

        // Pointers to data_col and data_im
        float* data_col_ptr = data_col + (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im + (channel_in * height + h_in) * width + w_in;

        // Nested loops unrolled
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;
                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                                data_im_ptr[i * width + j] : 0;
                data_col_ptr += height_col * width_col;
            }
        }
    }
}
```
