#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Ensure all threads are used before proceeding
    if(offset >= size) return;

    // Use shared memory to cache biases for effective memory access
    __shared__ float shared_bias;
    if(threadIdx.x == 0)
        shared_bias = biases[filter];
    __syncthreads();

    // Scale output by bias
    output[(batch*n+filter)*size + offset] *= shared_bias;
}