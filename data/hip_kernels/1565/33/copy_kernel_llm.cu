#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a stride loop to improve memory bandwidth utilization
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        Y[idx * INCY + OFFY] = X[idx * INCX + OFFX];
    }
}