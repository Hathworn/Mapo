#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX) 
{
    // Calculate the global index using block and grid dimensions
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use loop to process multiple elements per thread for better utilization
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x) 
    {
        X[idx * INCX] = ALPHA;
    }
}