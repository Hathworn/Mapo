#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure all threads are utilized properly by checking index range
    if(i < N) 
    {
        float x_val = X[i * INCX];  // Load input to register
        Y[i * INCY] = powf(x_val, ALPHA);  // Perform pow calculation and store the result
    }
}