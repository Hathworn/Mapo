#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Calculate the global index with 1D grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if within bounds
    if (i < n)
    {
        // Perform binarization
        binary[i] = (x[i] >= 0) ? 1 : -1;
    }
}