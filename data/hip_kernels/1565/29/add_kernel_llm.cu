#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use a single calculate for grid index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Avoid repeated computation
    if (i < N) {
        int index = i * INCX;
        X[index] += ALPHA;
    }
}