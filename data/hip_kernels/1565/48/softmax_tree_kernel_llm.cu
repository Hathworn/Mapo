#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    // Improved loop with parallel reduction for finding the largest value
    extern __shared__ float shared_data[];
    int tid = threadIdx.x;
    int index = tid * stride;

    shared_data[tid] = (index < n * stride) ? input[index] : -INFINITY;
    __syncthreads();

    // Parallel reduction to find max value
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_data[tid] = max(shared_data[tid], shared_data[tid + s]);
        }
        __syncthreads();
    }
    float largest = shared_data[0];
    
    // Compute softmax with improved memory read coalescing
    float sum = 0;
    if (index < n * stride) {
        float e = expf((input[index] - largest) / temp);
        atomicAdd(&sum, e);
        output[index] = e;
    }
    __syncthreads();

    // Normalize the output
    if (index < n * stride) {
        output[index] /= sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (id >= spatial * batch * groups) return;
    int s = id % spatial;
    id = id / spatial;
    int g = id % groups;
    int b = id / groups;
    int goff = group_offset[g] * spatial;
    int boff = b * stride;

    // Allocate shared memory size for max reduction
    size_t shared_mem_size = blockDim.x * sizeof(float);
    
    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}