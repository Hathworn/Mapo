#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if(i >= (NX+NY)*B) return;

    // Improved division and modulo calculations using shared variables
    int section_size = NX + NY;
    int b = i / section_size;
    int j = i % section_size;

    // Simplified conditions using early returns to reduce nesting
    if (j < NX) {
        // Check and update X
        if(X) {
            X[b*NX + j] += OUT[i];
        }
    } else {
        // Check and update Y
        if(Y) {
            Y[b*NY + (j - NX)] += OUT[i];
        }
    }
}