#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Improve index computation
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1 || f1 >= n || f2 >= n) return; // Include bounds checking

    float sum = 0.0f;
    float norm1 = 0.0f;
    float norm2 = 0.0f;

    // Use shared memory to cache frequently accessed data
    extern __shared__ float shared[]; 
    float* shared_output_f1 = shared;
    float* shared_output_f2 = shared + size;

    for (int b = 0; b < batch; ++b) {
        // Load data into shared memory
        for (int i = threadIdx.x; i < size; i += blockDim.x) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            shared_output_f1[i] = output[i1];
            shared_output_f2[i] = output[i2];
        }
        __syncthreads(); // Synchronize to ensure all data is loaded

        for (int i = 0; i < size; ++i) {
            sum += shared_output_f1[i] * shared_output_f2[i];
            norm1 += shared_output_f1[i] * shared_output_f1[i];
            norm2 += shared_output_f2[i] * shared_output_f2[i];
        }
        __syncthreads(); // Synchronize before using shared memory again
    }

    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    sum = sum / norm;

    for (int b = 0; b < batch; ++b) {
        for (int i = threadIdx.x; i < size; i += blockDim.x) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            float scaled_sum = - scale * sum / norm;
            atomicAdd(&delta[i1], scaled_sum * output[i2]);
            atomicAdd(&delta[i2], scaled_sum * output[i1]);
        }
    }
}