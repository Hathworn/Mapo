#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Efficient return if out of bounds
    if(i >= N) return;

    // Use lightweight calculations to determine indices
    int in_w = i % w;
    int in_h = (i / w) % h;
    int in_c = (i / (w * h)) % c;
    int b = (i / (w * h * c)) % batch;

    int out_c = c / (stride * stride);
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;

    int out_index = w2 + w * stride * (h2 + h * stride * (in_c % out_c + out_c * b));

    // Direct assignment based on the 'forward' flag
    if(forward) {
        out[out_index] = x[i];
    } else {
        out[i] = x[out_index];
    }
}