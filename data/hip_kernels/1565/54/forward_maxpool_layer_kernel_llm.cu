#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    int h = (in_h + 2*pad) / stride;
    int w = (in_w + 2*pad) / stride;
    int c = in_c;

    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int out_index = j + w * (i + h * (k + c * b));
    
    // Pre-compute the offsets for input coordinates
    int w_offset = -pad + j * stride;
    int h_offset = -pad + i * stride;

    float max = -INFINITY;
    int max_i = -1;

    // Use a single loop instead of nested loops
    for (int l = 0; l < size * size; ++l) {
        int cur_h = h_offset + (l / size);
        int cur_w = w_offset + (l % size);
        int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));

        int valid = (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w);
        float val = (valid != 0) ? input[index] : -INFINITY;
        if (val > max) {
            max = val;
            max_i = index;
        }
    }

    output[out_index] = max;
    indexes[out_index] = max_i;
}