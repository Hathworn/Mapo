#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Use shared memory for reduction within a block
    extern __shared__ float shared_variance_delta[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    shared_variance_delta[threadIdx.x] = 0;
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j*filters*spatial + i*spatial + k;
            shared_variance_delta[threadIdx.x] += delta[index] * (x[index] - mean[i]);
        }
    }
    
    // Synchronize threads within block for shared memory reduction
    __syncthreads();
    
    // Perform reduction using shared memory
    if (threadIdx.x == 0) {
        float block_sum = 0;
        for (int t = 0; t < blockDim.x; ++t) {
            block_sum += shared_variance_delta[t];
        }
        atomicAdd(&variance_delta[i], block_sum * -.5f * powf(variance[i] + .00001f, (float)(-3.f/2.f)));
    }
}