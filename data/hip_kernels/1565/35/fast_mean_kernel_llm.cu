#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int totalElements = batch * spatial;
    int offset = filter * spatial;
    
    // Unroll loop for efficiency and reduce bank conflicts
    for (int j = id; j < totalElements; j += threads) {
        int spatialIdx = j % spatial;
        if (spatialIdx < spatial) {
            int index = (j / spatial) * spatial * filters + offset + spatialIdx;
            local[id] += x[index];
        }
    }

    __syncthreads();

    // Reduce results using atomic technique
    if (id == 0) {
        float sum = 0.0f;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        mean[filter] = sum / totalElements;
    }
}