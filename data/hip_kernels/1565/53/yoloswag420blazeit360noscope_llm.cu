#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified calculation of id
    if (id < size) {
        float random_val = rand[id]; // Cache rand[id] to avoid repeated access
        if (random_val < prob) {
            input[id] = 0;
        } else {
            input[id] *= scale;
        }
    }
}