#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate global index, optimize for grid-stride loop
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use loop to cover all elements
    for (; i < n; i += stride) {
        c[i] += a[i] * b[i];
    }
}