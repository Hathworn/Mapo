#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;
    float sum = 0.0f; // Accumulator for each thread

    // Use a single loop with a stride of threads to reduce the number of iterations
    for (int j = 0; j < batch * spatial; j += threads) {
        int index = j * filters + filter * spatial + id;
        if (j + id < batch * spatial) { // Boundary check to prevent out-of-bounds access
            sum += delta[index];
        }
    }

    local[id] = sum; // Store each thread's sum in shared memory
    __syncthreads();

    // Reduce shared memory array to calculate the final mean_delta
    if (id == 0) {
        float total = 0.0f;
        for (int i = 0; i < threads; ++i) {
            total += local[i];
        }
        mean_delta[filter] = total * (-1.f / sqrtf(variance[filter] + .00001f));
    }
}