#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];
    
    int id = threadIdx.x;
    int filter = blockIdx.x;

    // Initialize shared memory for each thread
    local[id] = 0;

    int i, j;
    for(j = 0; j < batch; ++j) {
        // Loop unrolling for efficiency
        for(i = 0; i < spatial; i += threads * 4) {  // Unroll loop by processing 4 elements at once
            int index = j*spatial*filters + filter*spatial + i + id;
            if (i+id < spatial) {
                local[id] += delta[index]*(x[index] - mean[filter]);
            }
            if (i+id+threads < spatial) {
                local[id] += delta[index+threads]*(x[index+threads] - mean[filter]);
            }
            if (i+id+2*threads < spatial) {
                local[id] += delta[index+2*threads]*(x[index+2*threads] - mean[filter]);
            }
            if (i+id+3*threads < spatial) {
                local[id] += delta[index+3*threads]*(x[index+3*threads] - mean[filter]);
            }
        }
    }

    __syncthreads();

    // Use a single atomic operation to accumulate results in shared memory
    if (id == 0) {
        variance_delta[filter] += local[0];
        for(i = 1; i < threads; ++i) {
            variance_delta[filter] += local[i];
        }
        variance_delta[filter] *= -.5f * powf(variance[filter] + .00001f, (float)(-3.f/2.f));
    }
}