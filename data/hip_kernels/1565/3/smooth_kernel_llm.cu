#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    // Calculate the global thread ID
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Compute the multi-dimensional indices
    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    int w_offset = -(size / 2);
    int h_offset = -(size / 2);

    int out_index = j + w * (i + h * (k + c * b));
    float out_value = x[out_index];  // Store the output value for reuse

    for (int l = 0; l < size; ++l) {
        int cur_h = h_offset + i + l;
        if (cur_h >= 0 && cur_h < h) {  // Boundary check outside of inner loop
            for (int m = 0; m < size; ++m) {
                int cur_w = w_offset + j + m;
                if (cur_w >= 0 && cur_w < w) {  // Boundary check
                    int index = cur_w + w * (cur_h + h * (k + b * c));
                    delta[out_index] += rate * (x[index] - out_value);  // Use preloaded value
                }
            }
        }
    }
}