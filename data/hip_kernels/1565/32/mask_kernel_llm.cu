#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    // Calculate the global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Check bounds and mask condition
    if (i < n && mask[i] == mask_num) 
    {
        // Update the value if condition is true
        x[i] = mask_num;
    }
}