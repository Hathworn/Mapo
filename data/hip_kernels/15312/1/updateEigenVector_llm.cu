#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateEigenVector(float* d_b, float* d_temp, float* normAb, int n)
{
    // Compute unique index
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    // Load normAb once per block into shared memory
    __shared__ float sharedNormAb;
    if (threadIdx.x == 0) {
        sharedNormAb = *normAb;
    }
    __syncthreads();

    // Use stride within loop
    for (int i = index; i < n; i += blockDim.x * gridDim.x) {
        d_b[i] = d_temp[i] / sharedNormAb;
    }
}