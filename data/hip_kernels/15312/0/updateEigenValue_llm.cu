#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateEigenValue(int* d_rowPtr, int* d_col, float* d_val, float* d_b, float* d_temp, float* d_normAb, float* d_alpha1, float* d_alpha2, int n)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;
    float t1 = 0.0f;
    float t2 = 0.0f;
    float t3 = 0.0f;

    // Utilize fewer reads and writes by incrementing stride in a single loop
    for (int idx = index; idx < n; idx += stride) {
        float tempVal = 0.0f;
        for (int i = d_rowPtr[idx]; i < d_rowPtr[idx + 1]; i++) {
            tempVal += d_val[i] * d_b[d_col[i]];
        }
        d_temp[idx] = tempVal;
        // Accumulate dot products
        t1 += tempVal * tempVal;
        t2 += d_b[idx] * tempVal;
        t3 += d_b[idx] * d_b[idx];
    }

    // Use block-level reduction for accumulating dot product results
    __shared__ float cache1[2];
    __shared__ float cache2[2];
    __shared__ float cache3[2];

    // Initialize shared memory
    cache1[threadIdx.x] = t1;
    cache2[threadIdx.x] = t2;
    cache3[threadIdx.x] = t3;

    __syncthreads();

    // Perform reduction (use loop unrolling for better efficiency if blockDim.x is a power of 2)
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (threadIdx.x < i) {
            cache1[threadIdx.x] += cache1[threadIdx.x + i];
            cache2[threadIdx.x] += cache2[threadIdx.x + i];
            cache3[threadIdx.x] += cache3[threadIdx.x + i];
        }
        __syncthreads();
    }

    // Accumulate results with atomic operations outside of reduction loop
    if (threadIdx.x == 0) {
        atomicAdd(d_normAb, cache1[0]);
        atomicAdd(d_alpha1, cache2[0]);
        atomicAdd(d_alpha2, cache3[0]);
    }
}