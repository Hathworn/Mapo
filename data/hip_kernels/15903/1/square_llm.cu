#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(float* d_out, const float* __restrict__ d_in) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global index
    float f = d_in[idx];
    d_out[idx] = f * f;
}