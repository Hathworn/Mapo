#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cube(float* d_out, const float* __restrict__ d_in) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global thread index
    float f = d_in[idx];
    d_out[idx] = f * f * f;
}