#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use block and thread indices to identify each thread uniquely
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello world! I'm thread %d in block %d\n", globalThreadId, blockIdx.x);
}