#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define N 64

__global__ void addKernel(float *a, float *b) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce global memory access latency
    __shared__ float shared_a[blockDim.x];
    __shared__ float shared_b[blockDim.x];

    if (idx < N) {
        shared_a[threadIdx.x] = a[idx];
        shared_b[threadIdx.x] = b[idx];
        __syncthreads();

        // Perform addition in shared memory
        shared_a[threadIdx.x] += shared_b[threadIdx.x];
        __syncthreads();

        // Write the results back to global memory
        a[idx] = shared_a[threadIdx.x];
    }
}