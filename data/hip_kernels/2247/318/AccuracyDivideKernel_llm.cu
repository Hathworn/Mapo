#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Avoid dereferencing and dividing in a single step
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx == 0) {
        *accuracy /= N;
    }
}