#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BaseNeuronGetIntArray(int *arr1, int *arr2, int n_elem, int step1, int step2)
{
    // Calculate global index once for all threads in warp
    int array_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use warp-wide conditional check to improve branch performance
    if (array_idx < n_elem) {
        arr2[array_idx * step2] = arr1[array_idx * step1];
    }
}