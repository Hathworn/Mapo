#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BaseNeuronSetFloatPtArray(float *arr, int *pos, int n_elem, int step, float val)
{
    // Cache thread and block indices
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    // Calculate array index more efficiently
    int array_idx = tid + bid * blockDim.x;

    // Ensure we process elements within the range
    if (array_idx < n_elem) {
        // Directly update the element in arr using position and step offset
        arr[pos[array_idx] * step] = val;
    }
}