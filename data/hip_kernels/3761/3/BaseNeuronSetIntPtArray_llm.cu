#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BaseNeuronSetIntPtArray(int *arr, int *pos, int n_elem, int step, int val)
{
    // Compute the array index using built-in variables
    int array_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure that the thread is working within bounds
    if (array_idx < n_elem) {
        // Efficiently update the array value
        arr[pos[array_idx] * step] = val;
    }
}