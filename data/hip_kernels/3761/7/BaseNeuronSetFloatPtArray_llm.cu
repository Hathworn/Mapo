#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BaseNeuronSetFloatPtArray(float *arr, int *pos, int n_elem, int step, float val)
{
    // Use variable to avoid repeated calculations
    const int array_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure valid index before accessing memory
    if (array_idx < n_elem) {
        // Write value directly to calculated position
        arr[pos[array_idx] * step] = val;
    }
}