#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BaseNeuronSetIntArray(int *arr, int n_elem, int step, int val)
{
    // Calculate the global thread index
    int array_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure array index is within bounds
    if (array_idx < n_elem) {
        arr[array_idx * step] = val;
    }
}