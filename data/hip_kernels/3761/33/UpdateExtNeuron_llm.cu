#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UpdateExtNeuron(float *port_input_pt, float *port_value_pt, int n_node, int n_var, int n_port_var, int n_port)
{
    int i_thread = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Check if the thread index is within bounds
    if (i_thread < n_node * n_port) {
        int i_port = i_thread % n_port; // Calculate port index
        int i_node = i_thread / n_port; // Calculate node index

        // Compute pointer to the input port value
        float *pip = port_input_pt + i_node * n_var + n_port_var * i_port;

        // Update the port value and reset input port value
        port_value_pt[i_node * n_var + n_port_var * i_port] = *pip;
        *pip = 0.0f; // Set input port to zero
    }
}