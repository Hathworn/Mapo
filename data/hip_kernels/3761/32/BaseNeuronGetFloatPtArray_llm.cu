#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BaseNeuronGetFloatPtArray(float *arr1, float *arr2, int *pos, int n_elem, int step1, int step2)
{
    // Calculate the global index
    int array_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread index does not exceed the number of elements
    if (array_idx < n_elem) {
        // Use the index to update the output array
        arr2[array_idx * step2] = arr1[pos[array_idx] * step1];
    }
}