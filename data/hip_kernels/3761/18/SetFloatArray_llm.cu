#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SetFloatArray(float *arr, int n_elem, int step, float val)
{
    // Calculate unique global thread index
    int array_idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride to cover all elements in large arrays
    for (int i = array_idx; i < n_elem; i += gridDim.x * blockDim.x) {
        arr[i * step] = val;
    }
}