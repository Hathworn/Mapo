#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BaseNeuronSetIntPtArray(int *arr, int *pos, int n_elem, int step, int val)
{
    // Calculate the array index with cache of blockDim.x for repeated use
    int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Perform the operation only if global index is within bounds
    if (globalIdx < n_elem) {
        arr[pos[globalIdx] * step] = val;
    }
}