#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BaseNeuronGetFloatArray(float *arr1, float *arr2, int n_elem, int step1, int step2)
{
    int array_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process multiple elements per thread to improve memory coalescing and reduce kernel launch overhead
    int stride = blockDim.x * gridDim.x;
    for (int idx = array_idx; idx < n_elem; idx += stride) {
        arr2[idx * step2] = arr1[idx * step1];
    }
}