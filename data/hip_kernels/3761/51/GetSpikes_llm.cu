#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GetSpikes(double *spike_array, int array_size, int n_port, int n_var, float *port_weight_arr, int port_weight_arr_step, int port_weight_port_step, float *port_input_arr, int port_input_arr_step, int port_input_port_step)
{
    int i_array = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Avoid unnecessary calculations for out-of-bounds threads
    if (i_array >= array_size * n_port) return;
    
    int i_target = i_array % array_size;
    int port = i_array / array_size;
    int port_input = i_target * port_input_arr_step + port_input_port_step * port;
    int port_weight = i_target * port_weight_arr_step + port_weight_port_step * port;
    
    // Preloading values into registers
    float input_val = port_input_arr[port_input];
    double spike_val = (double)spike_array[i_array];
    float weight_val = port_weight_arr[port_weight];
    
    // Computing the result
    double d_val = spike_val + input_val * weight_val;
    
    // Storing the result back
    port_input_arr[port_input] = (float)d_val;
}