#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GetSpikes(double *spike_array, int array_size, int n_port, int n_var, float *port_weight_arr, int port_weight_arr_step, int port_weight_port_step, float *port_input_arr, int port_input_arr_step, int port_input_port_step)
{
    // Calculate global indices once to improve efficiency
    int i_target = blockIdx.x * blockDim.x + threadIdx.x;
    int port = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds to prevent out-of-bounds memory access
    if (i_target < array_size && port < n_port) {
        // Combine index calculations to reduce redundancy
        int i_array = port * array_size + i_target;
        int port_input = i_target * port_input_arr_step + port_input_port_step * port;
        int port_weight = i_target * port_weight_arr_step + port_weight_port_step * port;

        // Use registers directly to avoid repeated global memory access
        double input_value = (double)port_input_arr[port_input];
        double spike_value = spike_array[i_array];
        double weight_value = port_weight_arr[port_weight];

        // Perform computation using local variables
        double d_val = input_value + spike_value * weight_value;

        // Store result back to global memory
        port_input_arr[port_input] = (float)d_val;
    }
}