#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *output, int length, int *n1, int *n2) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    if (idx < length) {  // Ensure index within bounds
        output[idx] += n1[blockIdx.x] + n2[blockIdx.x];  // Perform operation
    }
}