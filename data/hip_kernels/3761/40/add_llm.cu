#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *output, int length, int *n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if (index < length * gridDim.x) { // Check bounds
        output[index] += n[blockIdx.x]; // Perform addition
    }
}