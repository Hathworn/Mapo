#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BaseNeuronSetIntArray(int *arr, int n_elem, int step, int val)
{
    int array_idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index

    if (array_idx < n_elem) { // Check if within bounds
        arr[array_idx * step] = val; // Set value with stride
    }
}