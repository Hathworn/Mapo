#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BaseNeuronSetFloatArray(float *arr, int n_elem, int step, float val)
{
    // Pre-compute values used multiple times to optimize performance
    int array_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = array_idx * step;

    // Check within bounds and set the array value
    if (array_idx < n_elem) {
        arr[offset] = val;
    }
}