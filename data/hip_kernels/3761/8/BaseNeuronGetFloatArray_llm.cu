#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BaseNeuronGetFloatArray(float *arr1, float *arr2, int n_elem, int step1, int step2)
{
    int array_idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll the loop to process multiple elements to improve performance
    int stride = blockDim.x * gridDim.x;
    for (int i = array_idx; i < n_elem; i += stride) {
        arr2[i * step2] = arr1[i * step1];
    }
}