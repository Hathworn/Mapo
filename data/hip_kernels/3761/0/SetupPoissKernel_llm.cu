#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SetupPoissKernel(hiprandState *curand_state, uint64_t n_dir_conn, unsigned long long seed)
{
    // Calculate the global thread index.
    uint64_t i_conn = (uint64_t)blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the index is within range.
    if (i_conn < n_dir_conn) {
        hiprand_init(seed, i_conn, 0, &curand_state[i_conn]);
    }
}