#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "SetupPoissKernel.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    hiprandState *curand_state = NULL;
hipMalloc(&curand_state, XSIZE*YSIZE);
uint64_t n_dir_conn = 1;
unsigned long long seed = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
SetupPoissKernel<<<gridBlock, threadBlock>>>(curand_state,n_dir_conn,seed);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
SetupPoissKernel<<<gridBlock, threadBlock>>>(curand_state,n_dir_conn,seed);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
SetupPoissKernel<<<gridBlock, threadBlock>>>(curand_state,n_dir_conn,seed);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}