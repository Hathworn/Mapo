#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BaseNeuronGetIntPtArray(int *arr1, int *arr2, int *pos, int n_elem, int step1, int step2)
{
    // Use shared memory for faster access
    __shared__ int shared_pos[256]; 
    int array_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (threadIdx.x < n_elem) {
        shared_pos[threadIdx.x] = pos[threadIdx.x];
    }
    __syncthreads(); // Ensure all threads have completed shared memory initialization

    if (array_idx < n_elem) {
        // Use shared memory to access 'pos' values
        arr2[array_idx * step2] = arr1[shared_pos[threadIdx.x] * step1];
    }
}