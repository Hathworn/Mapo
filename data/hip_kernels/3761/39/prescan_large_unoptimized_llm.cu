#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prescan_large_unoptimized(int *output, int *input, int n, int *sums) {
    int blockID = blockIdx.x;
    int threadID = threadIdx.x;
    int blockOffset = blockID * n;

    extern __shared__ int temp[];

    // Load input into shared memory with coalesced accesses
    temp[2 * threadID] = input[blockOffset + (2 * threadID)];
    temp[2 * threadID + 1] = input[blockOffset + (2 * threadID) + 1];

    int offset = 1;
    for (int d = n >> 1; d > 0; d >>= 1) {
        __syncthreads();
        if (threadID < d) {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            // Reduce redundant shared memory accesses
            int temp_ai = temp[ai];
            temp[bi] += temp_ai;
        }
        offset *= 2;
    }

    __syncthreads();

    // Directly set the last element to zero after storing sum
    if (threadID == 0) {
        sums[blockID] = temp[n - 1];
        temp[n - 1] = 0; 
    }

    for (int d = 1; d < n; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (threadID < d) {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            // Reduce redundant shared memory accesses
            int temp_ai = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += temp_ai;
        }
    }

    __syncthreads();

    // Store results globally with coalesced accesses
    output[blockOffset + (2 * threadID)] = temp[2 * threadID];
    output[blockOffset + (2 * threadID) + 1] = temp[2 * threadID + 1];
}