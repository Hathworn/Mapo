#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BaseNeuronGetIntArray(int *arr1, int *arr2, int n_elem, int step1, int step2)
{
    // Calculate array index for this thread
    int array_idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure array_idx is within bounds
    if (array_idx < n_elem) {
        // Efficiently assign value using calculated index
        arr2[array_idx * step2] = arr1[array_idx * step1];
    }
}