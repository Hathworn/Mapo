#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BaseNeuronGetFloatPtArray(float *arr1, float *arr2, int *pos, int n_elem, int step1, int step2)
{
    int array_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the current thread index is within bounds
    if (array_idx < n_elem) {
        // Directly calculate target index and assign value to avoid multiple indexing
        int target_idx = array_idx * step2;
        int source_idx = pos[array_idx] * step1;
        arr2[target_idx] = arr1[source_idx];
    }
}