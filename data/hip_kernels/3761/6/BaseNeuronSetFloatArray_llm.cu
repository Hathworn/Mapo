#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BaseNeuronSetFloatArray(float *arr, int n_elem, int step, float val)
{
    // Compute array index using unique thread index
    int array_idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if index is within bounds and update only valid elements
    if (array_idx < n_elem) {
        arr[array_idx * step] = val;
    }
}