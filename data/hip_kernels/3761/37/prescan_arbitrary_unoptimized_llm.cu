#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int locate(int val, int *data, int n)
{
    int i_left = 0;
    int i_right = n-1;
    int i = (i_left + i_right) / 2;
    while (i_right - i_left > 1) {
        if (data[i] > val) i_right = i;
        else if (data[i] < val) i_left = i;
        else break;
        i = (i_left + i_right) / 2;
    }
    return i;
}

__global__ void prescan_arbitrary_unoptimized(int *output, int *input, int n, int powerOfTwo) {
    extern __shared__ int temp[];
    int threadID = threadIdx.x;

    // Load input into shared memory with boundary check
    if (2 * threadID < n) {
        temp[2 * threadID] = input[2 * threadID];
        temp[2 * threadID + 1] = input[2 * threadID + 1];
    } else {
        temp[2 * threadID] = 0;
        temp[2 * threadID + 1] = 0;
    }

    int offset = 1;
    for (int d = powerOfTwo >> 1; d > 0; d >>= 1)
    {
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    // Clear the last element in the shared memory
    if (threadID == 0) { temp[powerOfTwo - 1] = 0; }

    for (int d = 1; d < powerOfTwo; d *= 2)
    {
        offset >>= 1;
        __syncthreads();
        if (threadID < d)
        {
            int ai = offset * (2 * threadID + 1) - 1;
            int bi = offset * (2 * threadID + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    // Write results to device memory with boundary check
    if (2 * threadID < n) {
        output[2 * threadID] = temp[2 * threadID];
        output[2 * threadID + 1] = temp[2 * threadID + 1];
    }
}