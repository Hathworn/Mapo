#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BaseNeuronGetIntPtArray(int *arr1, int *arr2, int *pos, int n_elem, int step1, int step2)
{
    int array_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (array_idx < n_elem) {
        int pos_idx = pos[array_idx]; // Cache position index for repeated access
        arr2[array_idx * step2] = arr1[pos_idx * step1];
    }
}