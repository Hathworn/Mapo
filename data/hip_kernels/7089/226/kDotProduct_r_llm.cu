#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    extern __shared__ float shmem[];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float temp = 0.0f; // Use register for reduction
    if (eidx < numElements) {
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            temp += a[eidx] * b[eidx];
        }
    }
    shmem[threadIdx.x] = temp;
    __syncthreads(); // Ensure all loads are complete

    // Reduce within shared memory using a loop to improve readability and future maintenance
    for (uint stride = DP_BLOCKSIZE / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads(); // Synchronize at each step of reduction
    }

    // Store the result for this block
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}