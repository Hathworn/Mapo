#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Early exit if most probable label isn't the true label
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            // Optimize numMax calculation using parallel reduction within a block
            int numMax = 0;
            for (int i = threadIdx.x; i < numOut; i += blockDim.x) {
                atomicAdd(&numMax, (probs[i * numCases + tx] == maxp) ? 1 : 0);
            }
            __shared__ int totalMax;
            if (threadIdx.x == 0) {
                totalMax = numMax;
                for (int i = 1; i < blockDim.x; i++) {
                    totalMax += __shfl_sync(0xFFFFFFFF, numMax, i);
                }
            }
            __syncthreads();
            if (threadIdx.x == 0) {
                correctProbs[tx] = 1.0f / float(totalMax);
            }
        }
    }
}