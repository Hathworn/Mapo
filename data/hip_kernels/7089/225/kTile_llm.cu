#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Utilize shared memory for loading source data to reduce global memory access
    extern __shared__ float sharedSrc[];

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    
    // Preload source data into shared memory
    for (uint i = threadIdx.x; i < srcWidth * srcHeight; i += blockDim.x) {
        sharedSrc[i] = src[i];
    }

    __syncthreads(); // Ensure all threads have loaded the data

    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        tgt[i] = sharedSrc[srcY * srcWidth + srcX];
    }
}