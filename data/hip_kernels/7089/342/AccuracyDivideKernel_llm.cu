#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Using atomic operation to ensure thread safety and accuracy
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        atomicExch(accuracy, *accuracy / N);
    }
}