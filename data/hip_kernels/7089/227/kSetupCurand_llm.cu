#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Initialize each thread with the seed and a unique sequence number
    hiprand_init(seed, tidx, 0, &state[tidx]);
}