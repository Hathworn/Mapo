#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    
    // Use warp-level primitives for faster reduction
    float tmp = 0.0f;
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    
    // Perform warp reduction
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        tmp += __shfl_down(tmp, offset);
    }
    
    // Store the result of warp reductions
    if (idx % warpSize == 0) {
        reduction_buffer[idx / warpSize] = tmp;
    }
    __syncthreads();
    
    // Further reduction across warps if necessary
    if (idx < blockDim.x / warpSize) {
        tmp = reduction_buffer[idx];
        for (int offset = blockDim.x / warpSize / 2; offset > 0; offset /= 2) {
            tmp += __shfl_down(tmp, offset);
        }
        
        if (idx == 0) {
            reduction_buffer[0] = tmp;
        }
    }
    __syncthreads();
    
    // Compute gradient based on reduced result
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}