#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < h && j < w) // Simplified boundary check
    {
        const int pos = i * s + j;
        float scale = normalization_factor[pos];
        float invScale = (scale != 0.0f) ? (1.0f / scale) : 1.0f; // Minor optimization on invScale calculation
        image[pos] *= invScale;
    }
}