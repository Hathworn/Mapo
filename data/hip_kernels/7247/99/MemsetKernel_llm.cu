#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global thread ID
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Use flattened index and bounds checking
    int pos = i * w + j;
    if (i < h && j < w) 
    {
        image[pos] = value; // Set value directly
    }
}