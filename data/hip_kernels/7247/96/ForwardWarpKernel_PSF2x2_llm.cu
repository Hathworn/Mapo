#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return; // Boundary check

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + static_cast<float>(i) + 1.0f;

    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);
    float value = src[image_row_offset + j];
    float weight;

    // Process bottom right corner
    if (tx < w && tx >= 0 && ty < h && ty >= 0) {
        weight = dx * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Process bottom left corner
    tx -= 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0) {
        weight = (1.0f - dx) * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Process upper left corner
    ty -= 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0) {
        weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Process upper right corner
    tx += 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0) {
        weight = dx * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}