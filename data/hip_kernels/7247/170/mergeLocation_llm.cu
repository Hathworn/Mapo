#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Use a single int to represent the global thread index
    int ptidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Compare once, store in temporary variable for further use
    if (ptidx < npoints) {
        // Load local memory only once for each value
        short lx = loc_[ptidx].x;
        short ly = loc_[ptidx].y;
        
        // Perform calculations and store directly in result arrays
        x[ptidx] = lx * scale;
        y[ptidx] = ly * scale;
    }
}