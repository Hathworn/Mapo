#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calc2points(float* point_coordinate_1, float* point_coordinate_2 , float* coordinates_arr)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // Use global thread index for scalability

    if (tid < blockDim.x) // Boundary check to prevent out-of-bound access
    {
        float diff = point_coordinate_1[tid] - point_coordinate_2[tid]; // Store result of subtraction
        coordinates_arr[tid] = diff * diff; // Avoid pow for better performance
    }
}