#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"
__global__ void cuExp(int n, float *a, float *result)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Iterate over data with a stride to use all available threads
    for (int index = i; index < n; index += blockDim.x * gridDim.x)
    {
        result[index] = exp(a[index]);
    }
}