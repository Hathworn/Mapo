#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

/*
* Perform a reduction from data of length 'size' to result, where length of result will be 'number of blocks'.
*/
extern "C"
__global__ void mult(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // Use registers to minimize memory access latency
        float valA = a[i];
        float valB = b[i];
        result[i] = valA * valB;
    }
}