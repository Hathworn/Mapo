#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel with more concise code
extern "C"
__global__ void cuLog(int n, float *a, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if index is within bounds
    if (i < n) {
        result[i] = logf(a[i]); // Use logf for better performance with float
    }
}