#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved version of the reduction kernel

/*
* Perform a reduction from data of length 'size' to result, where length of result will be 'number of blocks'.
*/
extern "C"
__global__ void reducePartial(int size, void *data, void *result) {
    float *fdata = (float*) data;
    float *sum = (float*) result;

    extern __shared__ float sdata[];

    // Calculate thread and block indices
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

    // Load data into shared memory and perform first level of reduction
    float mySum = (i < size ? fdata[i] : 0) + (i + blockDim.x < size ? fdata[i + blockDim.x] : 0);
    sdata[tid] = mySum;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) sum[blockIdx.x] = sdata[0];
}