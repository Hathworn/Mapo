#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
extern "C"
__global__ void accrue(int n, float *a, float *b, float p, float *result)
{
    // Compute unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure within bounds
    if (i < n)
    {
        result[i] = a[i] * (1.0f + b[i] * p);
    }
}