#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

extern "C"

/*
* Perform a reduction from data of length 'size' to result, 
* where length of result will be 'number of blocks'.
*/
extern "C"
__global__ void invert(int n, float *a, float *result)
{
    // Using shared memory to reduce global memory access latency
    __shared__ float shared_a[256];
    
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;

    if (index < n) {
        // Load data into shared memory
        shared_a[tid] = a[index];
        __syncthreads();
        
        // Perform inversion using shared memory
        result[index] = 1.0f / shared_a[tid];
    }
}