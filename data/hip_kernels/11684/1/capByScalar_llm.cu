#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Perform reduction with element-wise capping by scalar
extern "C"
__global__ void capByScalar(int n, float *a, float b, float *result)
{
    // Use shared memory to optimize memory access
    __shared__ float shared_cap;
    if (threadIdx.x == 0) shared_cap = b; // Load cap value to shared memory
    __syncthreads(); // Synchronize to ensure shared memory is ready

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // Access the shared cap for better performance
        result[i] = a[i] < shared_cap ? a[i] : shared_cap;
    }
}