#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to add scalar value 'b' to each element in array 'a'
extern "C"
__global__ void addScalar(int n, float *a, float b, float *result)
{
    // Optimize calculation of global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Efficient boundary check for threads working within the array bounds
    if (i < n)
    {
        // Perform vector addition computation
        result[i] = a[i] + b;
    }
}