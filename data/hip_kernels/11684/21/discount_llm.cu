#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
extern "C"
__global__ void discount(int n, float *a, float *b, float p, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure all threads execute calculations
    if (i < n)
    {
        result[i] = __fdividef(a[i], 1.0f + b[i] * p);  // Use fast intrinsic for division
    }
}