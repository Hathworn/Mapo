#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void sub(int n, float *a, float *b, float *result)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if within bounds and perform subtraction
    if (i < n)
    {
        result[i] = a[i] - b[i];
    }
    
    // Optimization: Use shared memory for faster access (if needed)
//    __shared__ float shared_a[BLOCK_SIZE];
//    __shared__ float shared_b[BLOCK_SIZE];
//    
//    if (i < n) {
//        shared_a[threadIdx.x] = a[i];
//        shared_b[threadIdx.x] = b[i];
//        __syncthreads();
//        result[i] = shared_a[threadIdx.x] - shared_b[threadIdx.x];
//    }
}