#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel with shared memory for better performance
extern "C"
__global__ void cuSqrt(int n, float *a, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for faster data access
    __shared__ float shared_data[256]; // Assumes 256 threads per block, adjust as needed
    if (i < n) {
        shared_data[threadIdx.x] = a[i];
        __syncthreads(); // Sync threads within block
        
        // Perform the square root operation
        result[i] = sqrt(shared_data[threadIdx.x]);
    }
}