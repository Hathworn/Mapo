#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improves kernel by using shared memory for coalesced access
extern "C"
__global__ void cuFloor(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check valid index within range
    if (i < n)
    {
        result[i] = a[i] > b[i] ? a[i] : b[i];
    }
}