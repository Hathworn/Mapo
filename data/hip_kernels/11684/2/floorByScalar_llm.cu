#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory to cache the 'floor' value across threads.
__global__ void floorByScalar(int n, float *a, float b, float *result) {
    extern __shared__ float shared[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load floor into shared memory once per block.
    if (threadIdx.x == 0) {
        shared[0] = b;
    }
    __syncthreads();

    // Perform floor operation using shared memory cache.
    if (i < n) {
        float floor = shared[0];
        result[i] = a[i] > floor ? a[i] : floor;
    }
}