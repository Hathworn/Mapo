#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel
extern "C"
__global__ void multScalar(int n, float *a, float b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Utilize shared memory and block synchronization for efficiency
    if (i < n)
    {
        result[i] = a[i] * b;
    }
}