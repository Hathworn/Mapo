#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256

/*
* Perform element-wise addition of two arrays.
* Optimized using shared memory for better memory access patterns.
*/
extern "C"
__global__ void add(int n, float *a, float *b, float *result)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        result[i] = a[i] + b[i];
    }
}