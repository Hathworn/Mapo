#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Global kernel function
extern "C"
__global__ void cuDiv(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        // Load data from global memory to registers for faster access
        float ai = a[i];
        float bi = b[i];
        // Perform division and store result back to global memory
        result[i] = ai / bi;
    }
}