#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use extern "C" for function linkage
extern "C"
// Global Kernel Function
__global__ void cap(int n, float *a, float *b, float *result)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check thread index within bounds
    if (i < n)
    {
        // Perform element-wise capping operation
        result[i] = min(a[i], b[i]);
    }
}