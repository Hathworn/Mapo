#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel using shared memory for coalesced memory access
extern "C" __global__ void addProduct_vs(int n, float *a, float *b, float c, float *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        result[i] = a[i] + b[i] * c; // Perform computation
    }
}