#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subScalar(int n, float *a, float b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by ensuring only valid threads perform operations
    if (i < n)
    {
        result[i] = a[i] - b;
    }
}