#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

/*
* Perform a reduction from data of length 'size' to result, where the length of result will be 'number of blocks'.
*/
__global__ void cuAbs(int n, float *a, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds
    if (i < n)
    {
        // Use fast math for absolute value
        result[i] = fabsf(a[i]);
    }
}