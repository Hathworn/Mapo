#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void add(int n, float *a, float *b, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // Use register for temporary storage to minimize global memory accesses
        float ai = a[i];
        float bi = b[i];
        sum[i] = ai + bi;
    }
}