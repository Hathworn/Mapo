#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function with shared memory for better performance
extern "C"
__global__ void squared(int n, float *a, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        // Use shared memory for faster data access
        __shared__ float shared_a[256];
        shared_a[threadIdx.x] = a[i];
        __syncthreads();

        // Perform computation using shared memory
        result[i] = shared_a[threadIdx.x] * shared_a[threadIdx.x];
    }
}