#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access and control divergence in the kernel function
extern "C"
__global__ void divScalar(int n, float *a, float b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // Use registers for temporary storage
        float ai = a[i];
        // Perform the division
        result[i] = ai / b;
    }
}