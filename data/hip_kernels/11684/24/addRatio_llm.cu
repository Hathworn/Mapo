#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Perform a reduction from data of length 'size' to result, where length of result will be 'number of blocks'.
*/
extern "C"
__global__ void addRatio(int n, float *a, float *b, float *c, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by checking if 'i' is within range before accessing arrays
    if (i < n)
    {
        // Perform addition and division in separate operations for better readability
        float divisionResult = b[i] / c[i];
        result[i] = a[i] + divisionResult;
    }
}