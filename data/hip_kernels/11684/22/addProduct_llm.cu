#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
extern "C"
__global__ void addProduct(int n, float *a, float *b, float *c, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // Use registers for intermediate computation to reduce memory access
        float bi = b[i];
        float ci = c[i];
        result[i] = a[i] + bi * ci;
    }
}