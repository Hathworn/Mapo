#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void cuPow(int n, float *a, float b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure every thread calculates within bounds
    if (i < n)
    {
        result[i] = powf(a[i], b); // Use powf for single precision
    }
}