#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void subRatio(int n, float *a, float *b, float *c, float *result)
{
    // Use a shared memory buffer for coalesced global memory access
    extern __shared__ float shared[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < n)
    {
        // Load data into shared memory
        shared[threadIdx.x] = b[i] / c[i];
        
        // Ensure all threads have loaded the data
        __syncthreads();
        
        // Perform operation using shared memory
        result[i] = a[i] - shared[threadIdx.x];
    }
}