#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

#define GAUSS_WIDTH 5
#define SOBEL_WIDTH 3

typedef struct images {
char *pType;
int width;
int height;
int maxValColor;
unsigned char *data;
} image;

__global__ void applyGaussianFilter(unsigned char *input, unsigned char *output, float *kernel, int iHeight, int iWidth, int kWidth) {

    // Calculate global position
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Ensure within image bounds
    if (x >= iWidth || y >= iHeight) return;

    double sum = 0.0;
    int halvedKW = kWidth / 2;

    // Optimization: Use constant memory if kernel size is fixed
    __shared__ float shKernel[GAUSS_WIDTH * GAUSS_WIDTH];

    if (threadIdx.x < kWidth && threadIdx.y < kWidth) {
        shKernel[threadIdx.y * kWidth + threadIdx.x] = kernel[threadIdx.y * kWidth + threadIdx.x];
    }
    __syncthreads();

    // Apply filter
    for (int i = -halvedKW; i <= halvedKW; i++) {
        for (int j = -halvedKW; j <= halvedKW; j++) {
            int nx = x + j;
            int ny = y + i;
            if (nx >= 0 && nx < iWidth && ny >= 0 && ny < iHeight) {
                int kPosX = j + halvedKW;
                int kPosY = i + halvedKW;
                sum += input[ny * iWidth + nx] * shKernel[kPosY * kWidth + kPosX];
            }
        }
    }

    // Clamp to max value
    if (sum > 255.0) sum = 255.0;

    // Write output
    output[y * iWidth + x] = (unsigned char)sum;
}