#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

#define GAUSS_WIDTH 5
#define SOBEL_WIDTH 3

typedef struct images {
    char *pType;
    int width;
    int height;
    int maxValColor;
    unsigned char *data;
} image;

__global__ void applySobelFilter(unsigned char *in, unsigned char *intensity, float *direction, int ih, int iw) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Check bounds once to avoid repeating checks
    if (x > 0 && x + 1 < iw && y > 0 && y + 1 < ih) {
        // Pre-compute common indices for efficiency
        int idx = y * iw + x;
        int idx_north = (y - 1) * iw;
        int idx_south = (y + 1) * iw;
        int idx_west = x - 1;
        int idx_east = x + 1;

        // Calculate gx and gy using pre-computed indices
        int gx = 
            in[idx_north + idx_west] - in[idx_north + idx_east] + 
            2 * (in[idx + idx_west] - in[idx + idx_east]) + 
            in[idx_south + idx_west] - in[idx_south + idx_east];

        int gy = 
            in[idx_north + idx_west] + 2 * in[idx_north + x] + in[idx_north + idx_east] - 
            in[idx_south + idx_west] - 2 * in[idx_south + x] - in[idx_south + idx_east];

        // Use single assignment statements for result calculations
        float gx_float = (float)gx;
        float gy_float = (float)gy;
        intensity[idx] = (unsigned char)sqrt(gx_float * gx_float + gy_float * gy_float);
        direction[idx] = atan2(gy_float, gx_float);
    }
}