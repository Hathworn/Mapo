#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel launch by setting up a unique state for each thread
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Unique thread index
    hiprand_init(seed, idx, 0, &state[idx]); // Initialize each thread's state
}