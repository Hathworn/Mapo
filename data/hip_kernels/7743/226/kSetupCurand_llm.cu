#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Each thread uses unique seed — improving randomness slightly
    hiprand_init(seed + tidx, tidx, 0, &state[tidx]);
}