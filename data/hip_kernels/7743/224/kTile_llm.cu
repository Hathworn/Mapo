#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate global thread ID
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    // Loop through elements with stride of total threads
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Calculate target coordinates and corresponding source coordinates
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        
        // Assign value from source to target using calculated coordinates
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}