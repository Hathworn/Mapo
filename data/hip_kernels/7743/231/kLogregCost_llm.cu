#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Use a shared memory counter for better performance on counting numMax
        __shared__ int sharedNumMax[LOGREG_ERR_THREADS_X];
        sharedNumMax[threadIdx.x] = 0;

        // Check if label probability is max probability
        if (labelp == maxp) {
            // Count how many probabilities equal to maxp
            for (int i = threadIdx.x; i < numOut; i += LOGREG_ERR_THREADS_X) {
                sharedNumMax[threadIdx.x] += probs[i * numCases + tx] == maxp;
            }
        }
        __syncthreads(); // Synchronize to ensure all threads have finished counting

        // Sum the counts in shared memory
        if (threadIdx.x == 0) {
            int numMax = 0;
            for (int i = 0; i < LOGREG_ERR_THREADS_X; i++) {
                numMax += sharedNumMax[i];
            }
            correctProbs[tx] = labelp == maxp ? 1.0f / float(numMax) : 0;
        }
    }
}