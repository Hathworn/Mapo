#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];

        // Perform multiplication with scale directly to save one register
        x[ptidx] = __fmul_rn(loc.x, scale);
        y[ptidx] = __fmul_rn(loc.y, scale);
    }
}