#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Pre-calculate repeatedly used variables to reduce operations
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py, dx, dy;
    dx = modff (cx, &px);
    dy = modff (cy, &py);
    int tx = (int) px;
    int ty = (int) py;
    float value = src[image_row_offset + j];

    float weight1 = dx * dy;
    float weight2 = (1.0f - dx) * dy;
    float weight3 = (1.0f - dx) * (1.0f - dy);
    float weight4 = dx * (1.0f - dy);

    // Combine repeated condition checks into single block to optimize branching
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        _atomicAdd(dst + ty * image_stride + tx, value * weight1);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight1);
    }

    tx -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        _atomicAdd(dst + ty * image_stride + tx, value * weight2);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight2);
    }

    ty -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        _atomicAdd(dst + ty * image_stride + tx, value * weight3);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight3);
    }

    tx += 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        _atomicAdd(dst + ty * image_stride + tx, value * weight4);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight4);
    }
}