#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global thread ID
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check for bounds and set value
    if (i < h && j < w)
    {
        image[i * w + j] = value;
    }
}