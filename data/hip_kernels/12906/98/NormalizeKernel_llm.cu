#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Early exit to avoid redundant calculations
    if (i >= h || j >= w) return;

    const int pos = i * s + j;

    float scale = normalization_factor[pos];

    // Use conditional operator for branchless computation
    float invScale = (scale != 0.0f) ? __fdividef(1.0f, scale) : 1.0f;

    image[pos] *= invScale; // Efficient multiplication
}