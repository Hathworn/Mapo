#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* __restrict__ loc_, float* __restrict__ x, float* __restrict__ y, const int npoints, const float scale)
{
    // Use a more efficient method for parallel execution
    int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Use register memory for temporary variables
        const short2 loc = loc_[ptidx];

        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}