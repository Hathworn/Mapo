#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate the 1D index of the current thread
    int index = blockIdx.y * blockDim.y * w + blockIdx.x * blockDim.x + threadIdx.y * w + threadIdx.x;

    // Calculate the maximum size for the 1D grid
    int maxIndex = w * h;

    // Check if the thread's index is within bounds
    if (index < maxIndex)
    {
        image[index] = value;
    }
}