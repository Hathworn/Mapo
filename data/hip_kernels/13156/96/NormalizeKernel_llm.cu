#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Compute unique global index for the thread
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIdy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the index is within bounds
    if (globalIdy < h && globalIdx < w) {
        const int pos = globalIdy * s + globalIdx;

        float scale = normalization_factor[pos];
        float invScale = (scale == 0.0f) ? 1.0f : __fdividef(1.0f, scale); // Use faster division

        image[pos] *= invScale; // Normalize the image pixel
    }
}