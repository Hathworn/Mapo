#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    // Consolidated flow and image row offsets
    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate bottom left corner of the target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);

    // Target pixel integer coordinates
    int tx = (int)(px);
    int ty = (int)(py);
    float value = src[image_row_offset + j];

    // Array to hold weights
    float weights[4] = {dx * dy, (1.0f - dx) * dy, (1.0f - dx) * (1.0f - dy), dx * (1.0f - dy)};
    int dx_offsets[4] = {0, -1, -1, 0};
    int dy_offsets[4] = {0, 0, -1, -1};

    for (int k = 0; k < 4; ++k) {
        int curr_tx = tx + dx_offsets[k];
        int curr_ty = ty + dy_offsets[k];

        // Check pixel bounds
        if (!(curr_tx >= w || curr_tx < 0 || curr_ty >= h || curr_ty < 0)) {
            // Atomic addition for pixel value and normalization factor
            _atomicAdd(dst + curr_ty * image_stride + curr_tx, value * weights[k]);
            _atomicAdd(normalization_factor + curr_ty * image_stride + curr_tx, weights[k]);
        }
    }
}