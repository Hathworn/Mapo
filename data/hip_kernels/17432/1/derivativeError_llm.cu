#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 32

__global__ void derivativeError(float *output, float *actual, float *deriv_err)
{
    __shared__ float sdata[BLOCK_SIZE]; // Adjust shared memory size for better usage

    // Calculate global index correctly for a 1D block configuration
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y;

    // Load data into shared memory
    sdata[threadIdx.x] = output[row * gridDim.y + col];
    __syncthreads();

    // Perform parallel reduction in shared memory
    for (int s = BLOCK_SIZE / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
            sdata[threadIdx.x] += sdata[threadIdx.x + s];
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0)
    {
        deriv_err[blockDim.y * blockIdx.x + col] = sdata[0];
    }
}