#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_distance(int* data, float* distance, int* point, int n, int dim) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (i < n) {
        float d = 0.0f;

        // Optimize by unrolling loops when possible
        for (int j = 0; j < dim; ++j) {
            d += fabsf(static_cast<float>(data[i * dim + j] - point[j])); // Use fabsf for float
        }

        distance[i] = d;
    }
}