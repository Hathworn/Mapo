#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(int* input, int* output) {
    __shared__ int tmp[TPB];

    // Load input into shared memory
    unsigned int tid = threadIdx.x;
    unsigned int index = tid + blockIdx.x * blockDim.x;
    tmp[tid] = input[index];

    __syncthreads();

    // Reduce using loop unrolling
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            tmp[tid] += tmp[tid + stride];
        }
        __syncthreads();
    }

    // Write result for this block to output
    if (tid == 0) {
        output[blockIdx.x] = tmp[0];
    }
}