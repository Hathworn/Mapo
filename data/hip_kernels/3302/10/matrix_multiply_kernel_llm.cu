#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    double out = 0.0;
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < dim_mn) {
        // Use shared memory to optimize global memory access
        extern __shared__ double shared_vector[];
        if (threadIdx.x < dim_mn) {
            shared_vector[threadIdx.x] = vector_in[threadIdx.x];
        }
        __syncthreads();

        // Unroll the loop to improve performance
        for (long j = 0; j < dim_mn; j += 4) {
            out += (j < dim_mn ? matrix[i * dim_mn + j] * shared_vector[j] : 0)
                 + (j + 1 < dim_mn ? matrix[i * dim_mn + j + 1] * shared_vector[j + 1] : 0)
                 + (j + 2 < dim_mn ? matrix[i * dim_mn + j + 2] * shared_vector[j + 2] : 0)
                 + (j + 3 < dim_mn ? matrix[i * dim_mn + j + 3] * shared_vector[j + 3] : 0);
        }

        vector_out[i] = out;
    }
}