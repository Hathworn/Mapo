#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bigstencil(int* in, int* out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensuring thread index is within bounds
    if (i < N) {
        out[i] = in[i] + 2;
    }
}