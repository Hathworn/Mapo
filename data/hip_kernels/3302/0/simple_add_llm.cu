#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_add(float* __restrict__ a, const float* __restrict__ b, int n)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use warp-synchronous programming when possible
    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x) {
        a[idx] += b[idx]; // Perform addition
    }
}