#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds once for both arrays
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];
        
        // Use registers for intermediate values
        float scaled_x = loc.x * scale;
        float scaled_y = loc.y * scale;

        x[ptidx] = scaled_x;
        y[ptidx] = scaled_y;
    }
}