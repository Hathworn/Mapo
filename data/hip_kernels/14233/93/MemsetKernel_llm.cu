#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Use a single index variable to minimize calculations
    int pos = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * blockIdx.y) + blockDim.x * blockDim.y * blockIdx.x;

    // Prevent index out of bounds using an early exit
    if (pos >= w * h) return;

    image[pos] = value;
}