#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global thread index
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread is within bounds
    if (i < h && j < w)
    {
        const int pos = i * s + j;
        
        // Use ternary operator for efficient scale inversion
        float scale = normalization_factor[pos];
        float invScale = (scale != 0.0f) ? __frcp_rn(scale) : 1.0f;

        // Apply normalization
        image[pos] *= invScale;
    }
}