#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Optimize pre-computation of common expressions
    float u_val = u[flow_row_offset + j] * time_scale;
    float v_val = v[flow_row_offset + j] * time_scale;
    float cx = u_val + static_cast<float>(j) + 1.0f;
    float cy = v_val + static_cast<float>(i) + 1.0f;

    float px, py;
    float dx = modff (cx, &px);
    float dy = modff (cy, &py);
    float value = src[image_row_offset + j];

    // Use inbounds function to reduce repeated checks
    auto inbounds = [&](int x, int y) -> bool { return (x >= 0 && x < w && y >= 0 && y < h); };

    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);

    // Pixel bottom right
    if (inbounds(tx, ty)) {
        float weight = dx * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Pixel bottom left
    tx -= 1;
    if (inbounds(tx, ty)) {
        float weight = (1.0f - dx) * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Pixel upper left
    ty -= 1;
    if (inbounds(tx, ty)) {
        float weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Pixel upper right
    tx += 1;
    if (inbounds(tx, ty)) {
        float weight = dx * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}