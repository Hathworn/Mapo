#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int blockSum(int *b, int size) {
    int sum = 0;
    for (int i = 0; i < size; ++i) {
        sum += b[i];
    }
    return sum;
}

__global__ void computeSumPerBlock(int *a, int N) {
    extern __shared__ int sdata_a[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        // Load element from global to shared memory
        sdata_a[tid] = a[i];
    } else {
        sdata_a[tid] = 0; // Ensures out-of-bounds access is safe
    }
    
    __syncthreads(); // Ensure shared memory is fully populated
    
    // Use reduction to sum values in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata_a[tid] += sdata_a[tid + s];
        }
        __syncthreads(); // Ensure all additions at this stage are done
    }
    
    // Write the result for this block to global memory
    if (tid == 0) {
        a[blockIdx.x] = sdata_a[0];
    }
}