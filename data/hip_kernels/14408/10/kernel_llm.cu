#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(int *A, int *B, int *counter, int n) {
    int tid = threadIdx.x;

    // Ensure shared memory is utilized to reduce global memory access time
    extern __shared__ int sharedCounter[];

    if (tid < n) {
        for (int j = 0; j < n; j++) {
            sharedCounter[tid * n + j] = counter[tid * n + j];
            sharedCounter[tid * n + j]++;  // Increment in shared memory
            A[tid * n + j] = B[tid * n + j];
            counter[tid * n + j] = sharedCounter[tid * n + j];  // Write back to global memory
        }
    }
}