#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(int *A, int *B, int *counter, int n) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    if (tid < n) {
        int baseIndex = tid * n; // Calculate base index once outside the loop
        for (int j = 0; j < n; j++) {
            int index = baseIndex + j;
            counter[index]++; // Reduce repeated calculation of index
            A[index] = B[index];
        }
    }
}