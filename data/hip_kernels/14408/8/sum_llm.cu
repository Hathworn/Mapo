#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using grid-stride loop
__global__ void sum(int *a, int *b, int *c, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < N; i += stride) {
        c[i] = a[i] + b[i];
    }
}