#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void Ring2_kernel(float *A, float *BP, int *corrAB, int *mask, int *m, int ring, int c, int h, int w) {
    // Assigns variables for thread and block indices
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;

    // Ensure operation is within the bounds of the input
    if (id1 < size) {
        // Check mask condition
        if (mask[id1] != 0) {
            // Load corrAB coordinates once to reduce redundant computations
            int y2 = corrAB[2 * id1 + 1];
            int x2 = corrAB[2 * id1];

            // Use shared memory to potentially optimize memory accesses
            __shared__ int shared_m[TB];

            // Iterate using a single loop for ring range
            for (int d = -ring; d <= ring; ++d) {
                int _x2 = x2 + d;
                int _y2 = y2 + d;

                // Check bounds
                if (_x2 >= 0 && _x2 < w) {
                    shared_m[threadIdx.x] = 1;
                    atomicExch(&m[_y2 * w + _x2], shared_m[threadIdx.x]);
                }
                if (_y2 >= 0 && _y2 < h) {
                    shared_m[threadIdx.x] = 1;
                    atomicExch(&m[_y2 * w + _x2], shared_m[threadIdx.x]);
                }
            }
        }
    }
    return;
}