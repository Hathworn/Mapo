#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void upsample_corr_kernel(int* curr_corr, int* next_corr, int curr_h, int curr_w, int next_h, int next_w)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < next_h * next_w) {
        int next_x = id % next_w, next_y = id / next_w;

        float w_ratio = (float)next_w / curr_w;
        float h_ratio = (float)next_h / curr_h;

        int curr_x = min(max(int((next_x + 0.5) / w_ratio), 0), curr_w - 1); // Clamp curr_x
        int curr_y = min(max(int((next_y + 0.5) / h_ratio), 0), curr_h - 1); // Clamp curr_y

        int curr_id = curr_y * curr_w + curr_x;
        int curr_x2 = curr_corr[2 * curr_id];
        int curr_y2 = curr_corr[2 * curr_id + 1];

        int next_x2 = min(max(int(next_x + (curr_x2 - curr_x) * w_ratio + 0.5), 0), next_w - 1); // Clamp next_x2
        int next_y2 = min(max(int(next_y + (curr_y2 - curr_y) * h_ratio + 0.5), 0), next_h - 1); // Clamp next_y2

        next_corr[2 * id] = next_x2;
        next_corr[2 * id + 1] = next_y2;
    }
}