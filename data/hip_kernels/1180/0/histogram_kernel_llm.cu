#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void histogram_kernel(float *I, float *minI, float *maxI, float *mask, int nbins, int c, int h, int w, float *hist)
{
    // Compute global thread ID
    int _id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;

    // Ensure the thread is within bounds
    if (_id >= c * size) return;

    int id = _id % size, dc = _id / size;

    // Check mask value
    if (mask[id] < EPS) return;

    float val = I[_id];
    float _minI = minI[dc];
    float _maxI = maxI[dc];

    // Adjust bounds in case min and max are equal
    if (_minI == _maxI) {
        _minI -= 1;
        _maxI += 1;
    }

    // Histogram binning
    if (_minI <= val && val <= _maxI) {
        int idx = MIN((val - _minI) / (_maxI - _minI) * nbins, nbins - 1);
        int index = dc * nbins + idx;
        // Use atomicAdd to avoid race conditions
        atomicAdd(&hist[index], 1.0f);
    }
}