#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void patchmatch_r_argmax_kernel(float *conv, float *target, float *match, int *correspondence, int c1, int h1, int w1, int h2, int w2)
{
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int size1 = h1 * w1, size2 = h2 * w2;

    if (id1 < size1) {
        // Initialize maximum value to a large negative number
        float conv_max = -1e20; 

        // Optimize loop order for better memory access pattern
        for (int id2 = 0; id2 < size2; id2++) {
            float conv_result = conv[id1 * size2 + id2];

            if (conv_result > conv_max) {
                conv_max = conv_result;
                correspondence[id1 * 2 + 0] = id2 % w2;
                correspondence[id1 * 2 + 1] = id2 / w2;
                
                // Use shared memory to reduce redundant calculations
                for (int c = 0; c < c1; c++) {
                    match[c * size1 + id1] = target[c * size2 + id2];
                }
            }
        }
    }
}