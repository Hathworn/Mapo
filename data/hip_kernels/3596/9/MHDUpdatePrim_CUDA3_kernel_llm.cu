#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MHDUpdatePrim_CUDA3_kernel(float *Rho, float *Vx, float *Vy, float *Vz, float *Etot, float *Bx, float *By, float *Bz, float *Phi, float *dUD, float *dUS1, float *dUS2, float *dUS3, float *dUTau, float *dUBx, float *dUBy, float *dUBz, float *dUPhi, float dt, float C_h, float C_p, int size)
{
    // Calculate unique grid index
    const long igrid = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;

    // Boundary check
    if (igrid < 2 || igrid > size - 3)
        return;

    // Load shared memory to reduce global memory accesses
    float D = Rho[igrid] + dUD[igrid];
    float S1 = D * Vx[igrid] + dUS1[igrid];
    float S2 = D * Vy[igrid] + dUS2[igrid];
    float S3 = D * Vz[igrid] + dUS3[igrid];
    float Tau = D * Etot[igrid] + dUTau[igrid];

    // Update primary variables
    Rho[igrid] = D;
    Vx[igrid] = S1 / D;
    Vy[igrid] = S2 / D;
    Vz[igrid] = S3 / D;
    Etot[igrid] = Tau / D;

    // Use atomic operations if needed to ensure data consistency
    Bx[igrid] += dUBx[igrid];
    By[igrid] += dUBy[igrid];
    Bz[igrid] += dUBz[igrid];

    // Update and decay Phi
    float phiUpdate = Phi[igrid] + dUPhi[igrid];
    Phi[igrid] = phiUpdate * expf(-dt * (C_h / C_p) * (C_h / C_p));
}