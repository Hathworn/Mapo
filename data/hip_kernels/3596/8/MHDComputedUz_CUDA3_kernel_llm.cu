#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MHDComputedUz_CUDA3_kernel(float *FluxD, float *FluxS1, float *FluxS2, float *FluxS3, float *FluxTau, float *FluxBx, float *FluxBy, float *FluxBz, float *FluxPhi, float *dUD, float *dUS1, float *dUS2, float *dUS3, float *dUTau, float *dUBx, float *dUBy, float *dUBz, float *dUPhi, float dtdx, int size, int dim0, int dim1, int dim2)
{
    // get thread and block index
    const long tx = threadIdx.x;
    const long bx = blockIdx.x;
    const long by = blockIdx.y;

    int igridz = tx + bx * CUDA_BLOCK_SIZE + by * CUDA_BLOCK_SIZE * CUDA_GRID_SIZE;

    // early exit for out-of-bound threads
    if (igridz < 2 || igridz > size - 3) return;

    int j = igridz / (dim0 * dim2);
    int i = (igridz - j * dim0 * dim2) / dim2;
    int k = igridz - j * dim0 * dim2 - i * dim2;
    int igrid = i + (j + k * dim1) * dim0;
    
    // compute position for the next point in the z-direction
    int igridzp1 = igridz + 1;
    int jp1 = igridzp1 / (dim0 * dim2);
    int ip1 = (igridzp1 - jp1 * dim0 * dim2) / dim2;
    int kp1 = igridzp1 - jp1 * dim0 * dim2 - ip1 * dim2;
    int igridp1 = ip1 + (jp1 + kp1 * dim1) * dim0;

    // use shared calculation results
    float dtdx_res = dtdx;
    float diffD = FluxD[igrid] - FluxD[igridp1];
    float diffS1 = FluxS1[igrid] - FluxS1[igridp1];
    float diffS2 = FluxS2[igrid] - FluxS2[igridp1];
    float diffS3 = FluxS3[igrid] - FluxS3[igridp1];
    float diffTau = FluxTau[igrid] - FluxTau[igridp1];
    float diffBx = FluxBx[igrid] - FluxBx[igridp1];
    float diffBy = FluxBy[igrid] - FluxBy[igridp1];
    float diffBz = FluxBz[igrid] - FluxBz[igridp1];
    float diffPhi = FluxPhi[igrid] - FluxPhi[igridp1];

    // update the arrays
    dUD[igrid] += diffD * dtdx_res;
    dUS1[igrid] += diffS1 * dtdx_res;
    dUS2[igrid] += diffS2 * dtdx_res;
    dUS3[igrid] += diffS3 * dtdx_res;
    dUTau[igrid] += diffTau * dtdx_res;
    dUBx[igrid] += diffBx * dtdx_res;
    dUBy[igrid] += diffBy * dtdx_res;
    dUBz[igrid] += diffBz * dtdx_res;
    dUPhi[igrid] += diffPhi * dtdx_res;
}