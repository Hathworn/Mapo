#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define CUDA_BLOCK_SIZE 256 // Assuming these constants are defined
#define CUDA_GRID_SIZE 256 

__global__ void HydroComputedUx_CUDA3_kernel(float *FluxD, float *FluxS1, float *FluxS2, float *FluxS3, float *FluxTau, float *dUD, float *dUS1, float *dUS2, float *dUS3, float *dUTau, float dtdx, int size)
{
    // Get global thread index using built-in functions
    const long igrid = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Process only valid indices to avoid wasted computations
    if (igrid >= 2 && igrid <= size - 3) {
        int igridp1 = igrid + 1;

        // Efficient memory access by coalesced reads/writes
        dUD  [igrid] = (FluxD  [igrid] - FluxD  [igridp1]) * dtdx;
        dUS1 [igrid] = (FluxS1 [igrid] - FluxS1 [igridp1]) * dtdx;
        dUS2 [igrid] = (FluxS2 [igrid] - FluxS2 [igridp1]) * dtdx;
        dUS3 [igrid] = (FluxS3 [igrid] - FluxS3 [igridp1]) * dtdx;
        dUTau[igrid] = (FluxTau[igrid] - FluxTau[igridp1]) * dtdx;
    }
}