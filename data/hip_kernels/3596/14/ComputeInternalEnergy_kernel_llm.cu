#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeInternalEnergy_kernel(float *Vx, float *Vy, float *Vz, float *Etot, float *Eneint, int size)
{
    // Get global thread index directly using blockDim.x and blockIdx.x/y
    int igrid = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;

    // Ensure thread does not process out-of-bounds memory
    if (igrid >= size) return;

    // Calculate kinetic energy and internal energy
    float velocitySquared = Vx[igrid] * Vx[igrid] + Vy[igrid] * Vy[igrid] + Vz[igrid] * Vz[igrid];
    Eneint[igrid] = Etot[igrid] - 0.5f * velocitySquared;
}