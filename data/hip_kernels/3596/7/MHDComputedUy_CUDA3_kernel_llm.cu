#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MHDComputedUy_CUDA3_kernel(float *FluxD, float *FluxS1, float *FluxS2, float *FluxS3, float *FluxTau, float *FluxBx, float *FluxBy, float *FluxBz, float *FluxPhi, float *dUD, float *dUS1, float *dUS2, float *dUS3, float *dUTau, float *dUBx, float *dUBy, float *dUBz, float *dUPhi, float dtdx, int size, int dim0, int dim1, int dim2)
{
    // Get flattened global thread ID
    const int igridy = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;

    // Thread boundary check
    if (igridy < 2 || igridy > size - 3)
        return;

    // Compute grid indices using less operations
    int temp = igridy;
    const int k = temp / (dim0 * dim1);
    temp -= k * dim0 * dim1;
    const int i = temp / dim1;
    const int j = temp % dim1;
    const int igrid = i + (j + k * dim1) * dim0;

    const int igridyp1 = igridy + 1;
    temp = igridyp1;
    const int kp1 = temp / (dim0 * dim1);
    temp -= kp1 * dim0 * dim1;
    const int ip1 = temp / dim1;
    const int jp1 = temp % dim1;
    const int igridp1 = ip1 + (jp1 + kp1 * dim1) * dim0;

    // Update dU vectors in a loop to reduce redundancy
    float *fluxArr[9] = {FluxD, FluxS1, FluxS2, FluxS3, FluxTau, FluxBx, FluxBy, FluxBz, FluxPhi};
    float *dUArr[9] = {dUD, dUS1, dUS2, dUS3, dUTau, dUBx, dUBy, dUBz, dUPhi};

    for (int idx = 0; idx < 9; ++idx) {
        dUArr[idx][igrid] += (fluxArr[idx][igrid] - fluxArr[idx][igridp1]) * dtdx;
    }
}