#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void HydroUpdatePrim_CUDA3_kernel(float *Rho, float *Vx, float *Vy, float *Vz, float *Etot, float *dUD, float *dUS1, float *dUS2, float *dUS3, float *dUTau, float dt, int size)
{
    // Simplified thread index calculation using intrinsic functions
    int igrid = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Early return for threads not in valid range
    if (igrid < 2 || igrid >= size - 2) {
        return;
    }

    // Reorganize data access for better memory coalescing
    float D   = Rho[igrid] + dUD[igrid];
    float S1  = D * Vx[igrid] + dUS1[igrid];
    float S2  = D * Vy[igrid] + dUS2[igrid];
    float S3  = D * Vz[igrid] + dUS3[igrid];
    float Tau = D * Etot[igrid] + dUTau[igrid];

    // Apply updates
    Rho[igrid]  = D;
    Vx[igrid]   = S1 / D;
    Vy[igrid]   = S2 / D;
    Vz[igrid]   = S3 / D;
    Etot[igrid] = Tau / D;
}