#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeInternalEnergy_kernel(float *Rho, float *Vx, float *Vy, float *Vz, float *Etot, float *Eneint, float *Bx, float *By, float *Bz, int size)
{
    // calculate global thread index
    const long igrid = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;

    if (igrid < size) {
        // optimize internal energy computation by minimizing redundant calculations
        const float velocity_squared = Vx[igrid] * Vx[igrid] + Vy[igrid] * Vy[igrid] + Vz[igrid] * Vz[igrid];
        const float magnetic_squared = (Bx[igrid] * Bx[igrid] + By[igrid] * By[igrid] + Bz[igrid] * Bz[igrid]) / Rho[igrid];

        Eneint[igrid] = Etot[igrid] - 0.5f * (velocity_squared + magnetic_squared);
    }
}