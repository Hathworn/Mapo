#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Define constants for better readability and potential reuse
#define THREADS_PER_BLOCK CUDA_BLOCK_SIZE
#define GRID_SIZE CUDA_GRID_SIZE

__global__ void HydroComputedUz_CUDA3_kernel(float *FluxD, float *FluxS1, float *FluxS2, float *FluxS3, float *FluxTau, float *dUD, float *dUS1, float *dUS2, float *dUS3, float *dUTau, float dtdx, int size, int dim0, int dim1, int dim2)
{
    // Calculate global index
    const long igridz = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK + blockIdx.y * THREADS_PER_BLOCK * GRID_SIZE;

    if (igridz >= 2 && igridz <= size - 3)
    {
        // Precompute reused values
        const int dim0_dim2 = dim0 * dim2;
        const int j = igridz / dim0_dim2;
        const int temp = igridz - j * dim0_dim2;
        const int i = temp / dim2;
        const int k = temp - i * dim2;
        const int igrid = i + (j + k * dim1) * dim0;

        const int igridzp1 = igridz + 1;
        const int j_p1 = igridzp1 / dim0_dim2;
        const int temp_p1 = igridzp1 - j_p1 * dim0_dim2;
        const int i_p1 = temp_p1 / dim2;
        const int k_p1 = temp_p1 - i_p1 * dim2;
        const int igridp1 = i_p1 + (j_p1 + k_p1 * dim1) * dim0;

        // Perform the computation
        const float multiplier = dtdx;
        dUD[igrid] += (FluxD[igrid] - FluxD[igridp1]) * multiplier;
        dUS1[igrid] += (FluxS1[igrid] - FluxS1[igridp1]) * multiplier;
        dUS2[igrid] += (FluxS2[igrid] - FluxS2[igridp1]) * multiplier;
        dUS3[igrid] += (FluxS3[igrid] - FluxS3[igridp1]) * multiplier;
        dUTau[igrid] += (FluxTau[igrid] - FluxTau[igridp1]) * multiplier;
    }
}