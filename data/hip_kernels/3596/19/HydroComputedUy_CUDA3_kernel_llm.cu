#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void HydroComputedUy_CUDA3_kernel(float *FluxD, float *FluxS1, float *FluxS2, float *FluxS3, float *FluxTau, float *dUD, float *dUS1, float *dUS2, float *dUS3, float *dUTau, float dtdx, int size, int dim0, int dim1, int dim2)
{
    // Calculate linear index
    const long tx = threadIdx.x;
    const long bx = blockIdx.x;
    const long by = blockIdx.y;
    int igridy = tx + bx * CUDA_BLOCK_SIZE + by * CUDA_BLOCK_SIZE * CUDA_GRID_SIZE;

    if (igridy < 2 || igridy > size - 3) return;

    // Calculate i, j, k from igridy
    int k = igridy / (dim0 * dim1);
    int rem = igridy - k * (dim0 * dim1);
    int i = rem / dim1;
    int j = rem % dim1;

    int igrid = i + (j + k * dim1) * dim0;

    // Reduce redundant calculation by reusing variables
    int igridyp1 = igridy + 1;
    k = igridyp1 / (dim0 * dim1);
    rem = igridyp1 - k * (dim0 * dim1);
    i = rem / dim1;
    j = rem % dim1;
    int igridp1 = i + (j + k * dim1) * dim0;

    // Update dU arrays using shared expressions
    float diffD = (FluxD[igrid] - FluxD[igridp1]) * dtdx;
    float diffS1 = (FluxS1[igrid] - FluxS1[igridp1]) * dtdx;
    float diffS2 = (FluxS2[igrid] - FluxS2[igridp1]) * dtdx;
    float diffS3 = (FluxS3[igrid] - FluxS3[igridp1]) * dtdx;
    float diffTau = (FluxTau[igrid] - FluxTau[igridp1]) * dtdx;

    dUD[igrid] += diffD;
    dUS1[igrid] += diffS1;
    dUS2[igrid] += diffS2;
    dUS3[igrid] += diffS3;
    dUTau[igrid] += diffTau;
}