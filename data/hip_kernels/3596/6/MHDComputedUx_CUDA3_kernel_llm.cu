#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MHDComputedUx_CUDA3_kernel(float *FluxD, float *FluxS1, float *FluxS2, float *FluxS3, float *FluxTau, float *FluxBx, float *FluxBy, float *FluxBz, float *FluxPhi, float *dUD, float *dUS1, float *dUS2, float *dUS3, float *dUTau, float *dUBx, float *dUBy, float *dUBz, float *dUPhi, float dtdx, int size)
{
    // Calculate global grid index
    int igrid = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;

    // Boundary check
    if (igrid < 2 || igrid > size - 3)
        return;

    // Pre-calculate index
    int igridp1 = igrid + 1;

    // Use pointers for direct memory access to reduce overhead
    float diff_dtdx = dtdx;

    // Perform computations with reduced register pressure
    dUD[igrid] = (FluxD[igrid] - FluxD[igridp1]) * diff_dtdx;
    dUS1[igrid] = (FluxS1[igrid] - FluxS1[igridp1]) * diff_dtdx;
    dUS2[igrid] = (FluxS2[igrid] - FluxS2[igridp1]) * diff_dtdx;
    dUS3[igrid] = (FluxS3[igrid] - FluxS3[igridp1]) * diff_dtdx;
    dUTau[igrid] = (FluxTau[igrid] - FluxTau[igridp1]) * diff_dtdx;
    dUBx[igrid] = (FluxBx[igrid] - FluxBx[igridp1]) * diff_dtdx;
    dUBy[igrid] = (FluxBy[igrid] - FluxBy[igridp1]) * diff_dtdx;
    dUBz[igrid] = (FluxBz[igrid] - FluxBz[igridp1]) * diff_dtdx;
    dUPhi[igrid] = (FluxPhi[igrid] - FluxPhi[igridp1]) * diff_dtdx;
}