#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function
__global__ void square(int *array, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (tid < n) { // Boundary check
        array[tid] *= array[tid]; // Square the element
    }
}