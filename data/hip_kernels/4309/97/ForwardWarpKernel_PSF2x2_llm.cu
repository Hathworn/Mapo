#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Precompute cx, cy, dx, dy, and value
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);
    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];

    // Merge condition checks and atomic adds
    #pragma unroll 4
    for (int offset_x = 0; offset_x <= 1; ++offset_x)
    {
        for (int offset_y = 0; offset_y <= 1; ++offset_y)
        {
            int adj_tx = tx - offset_x;
            int adj_ty = ty - offset_y;
            if (adj_tx >= 0 && adj_tx < w && adj_ty >= 0 && adj_ty < h)
            {
                float weight = (offset_x ? dx : 1.0f - dx) * (offset_y ? dy : 1.0f - dy);
                atomicAdd(dst + adj_ty * image_stride + adj_tx, value * weight);
                atomicAdd(normalization_factor + adj_ty * image_stride + adj_tx, weight);
            }
        }
    }
}