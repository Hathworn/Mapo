#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate 2D index
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Early return if out of bounds
    if (i >= h || j >= w) return;

    // Compute linear position
    const int pos = i * s + j;

    // Load normalization factor
    float scale = normalization_factor[pos];

    // Inverse scale using ternary operator for clarity and efficiency
    float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

    // Scale the image value
    image[pos] *= invScale;
}