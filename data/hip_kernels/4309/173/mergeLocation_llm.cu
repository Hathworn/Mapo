#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate point index
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Utilize shared memory to reduce global memory accesses
    __shared__ short2 shared_loc[256]; // Assuming blockDim.x <= 256

    if (ptidx < npoints)
    {
        // Load data into shared memory
        shared_loc[threadIdx.x] = loc_[ptidx];
        __syncthreads(); // Ensure all threads have loaded their data

        // Perform computation using shared memory
        short2 loc = shared_loc[threadIdx.x];
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}