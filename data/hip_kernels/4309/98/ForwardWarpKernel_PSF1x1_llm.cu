#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst) {
    // Calculate global indices
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Return if the indices are out of bounds
    if (i >= h || j >= w) return;

    // Calculate offsets
    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Read flow values
    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Compute target pixel's bottom left corner
    float cx = u_ * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v_ * time_scale + static_cast<float>(i) + 1.0f;

    // Determine the target pixel integer position
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Read source value
    float value = src[image_row_offset + j];

    // Check boundaries and perform atomic add
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0)) {
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}
```
