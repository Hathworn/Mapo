#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    const float *Ar = A_in + blockIdx.x * ldi;
    float *Aw = A_out + blockIdx.x * ldo;

    int half_length = row_length / 2;
    for (int idx = thread_id; idx < half_length; idx += gridDim.x * blockDim.x) {
        // Ignore special case where idx is 0
        if (idx == 0) continue;

        float2 val = reinterpret_cast<const float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret); // Assign directly using ternary operator for optimization
    }

    // Handle special case using first thread of a block
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];
        Aw[0] = use_power ? real * real : fabs(real); // Simplify condition using ternary operator
        Aw[half_length] = use_power ? im * im : fabs(im); // Simplify condition using ternary operator
    }
}