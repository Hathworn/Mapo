#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_linear_and_quadratic_terms_kernel( int32_t n, float old_num_frames, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {

    // Pre-calculate values that don't change across threads
    float cur_weight = *cur_tot_weight;
    float new_num_frames = old_num_frames + cur_weight;
    float prior_scale_change = 1.0f;

    if (max_count != 0) {
        // Use reciprocal to avoid repeated division
        float inv_max_count = 1.0f / max_count; 
        float old_prior_scale = max(old_num_frames, (float)max_count) * inv_max_count;
        float new_prior_scale = max(new_num_frames, (float)max_count) * inv_max_count;
        prior_scale_change += new_prior_scale - old_prior_scale;
    }

    // Optimize loop stride for better performance
    for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        quadratic[diag_idx] += prior_scale_change;
    }

    // Separate block for thread 0 to avoid unnecessary if-checks in loop
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        linear[0] += prior_offset * prior_scale_change;
    }
}