#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    
    // Efficient shared memory usage
    __shared__ float shared_Ar[CU1DBLOCK * 2];

    // Load data into shared memory
    for (int idx = thread_id * 2; idx < row_length; idx += CU1DBLOCK * 2) {
        shared_Ar[idx] = Ar[idx];
        if (idx + 1 < row_length) {
            shared_Ar[idx + 1] = Ar[idx + 1];
        }
    }
    __syncthreads();

    for (int idx = thread_id; idx < half_length; idx += CU1DBLOCK) {
        if (idx == 0) continue;

        float2 val = reinterpret_cast<float2 *>(shared_Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        if (use_power) {
            Aw[idx] = ret;
        } else {
            Aw[idx] = sqrtf(ret);
        }
    }

    // Streamline special case handling
    if (thread_id == 0) {
        float real = shared_Ar[0];
        float im = shared_Ar[row_length - 1];

        if (use_power) {
            Aw[0] = real * real;
            Aw[half_length] = im * im;
        } else {
            Aw[0] = fabs(real);
            Aw[half_length] = fabs(im);
        }
    }
}