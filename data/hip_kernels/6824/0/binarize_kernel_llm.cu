#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation for 1D grid
    if (i < n) { // Keep condition for valid index range check
        binary[i] = (x[i] >= 0) ? 1.0f : -1.0f; // Ensure consistent float assignment
    }
}