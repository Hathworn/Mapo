#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Compute linear index for 1D grid
    if(i < N)
    {
        int idxX = i * INCX; // Compute index for X
        int idxY = i * INCY; // Compute index for Y
        Y[idxY] = powf(X[idxX], ALPHA); // Use powf for float
    }
}