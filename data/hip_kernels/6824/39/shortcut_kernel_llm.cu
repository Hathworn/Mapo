#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    // Calculate unique thread id
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride of grid to loop over all elements
    for (; id < size; id += gridDim.x * blockDim.x) {
        int i = id % minw;
        int j = (id / minw) % minh;
        int k = (id / (minw * minh)) % minc;
        int b = (id / (minw * minh * minc)) % batch;

        // Compute indices
        int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
        int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));
        
        // Perform operation
        out[out_index] += add[add_index];
    }
}