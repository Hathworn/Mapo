#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Use shared memory for mean to reduce global memory access
    __shared__ float temp[1024]; 
    float sum = 0.0f;

    // Each thread calculates a part of the mean value
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        sum += fabsf(weights[f * size + i]);
    }
    temp[threadIdx.x] = sum;

    // Reduce within the block to get the final mean
    __syncthreads();
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            temp[threadIdx.x] += temp[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Calculate mean using first thread of the block
    if (threadIdx.x == 0) {
        temp[0] = temp[0] / size;
    }
    __syncthreads();

    // Assign binary weights based on mean
    float mean = temp[0];
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    }
}