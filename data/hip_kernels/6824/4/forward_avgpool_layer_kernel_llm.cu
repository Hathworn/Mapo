#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    // Calculate unique global thread ID.
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n * c) return;

    // Calculate batch and channel indexes.
    int b = id / c;
    int k = id % c;
    int out_index = k + c * b;

    // Initialize sum to zero.
    float sum = 0.0f;

    // Use local variable to accumulate sum for better performance.
    for (int i = 0; i < w * h; ++i) {
        int in_index = i + (w * h * (k + b * c));
        sum += input[in_index];
    }

    // Store the averaged result in output.
    output[out_index] = sum / (w * h);
}