#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop over the grid with stride to cover all elements
    for (int i = idx; i < N; i += blockDim.x * gridDim.x) 
    {
        X[i*INCX] = ALPHA;
    }
}