#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Optimize thread index calculation
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process in steps of total threads
    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x) {
        float diff = truth[idx] - pred[idx];
        error[idx] = fabsf(diff); // Use faster intrinsic function
        delta[idx] = (diff > 0.0f) ? 1.0f : -1.0f; // Ensure type consistency
    }
}