#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function using grid-stride loop for better performance
__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(; i < N; i += stride) {
        X[i*INCX] *= ALPHA;
    }
}