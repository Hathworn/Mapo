#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop to handle larger data sets and increase GPU utilization
    for (; index < N; index += blockDim.x * gridDim.x) {
        int f = (index / spatial) % filters;
        x[index] = (x[index] - mean[f]) / (sqrtf(variance[f] + .00001f));
    }
}