#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Reduce redundant recalculations of common expressions
    int part_size = BLOCK * size;
    int filter_size = filter + n * batch;

    for(int b = 0; b < batch; ++b){
        for(int i = p; i < size; i += BLOCK){
            int index = i + size * filter_size + size * b;
            // Ensure that only valid indices are accessed
            if (i < size) {
                sum += delta[index] * x_norm[index];
            }
        }
    }
    
    part[p] = sum;
    __syncthreads();

    // Use parallel reduction for better performance
    if (p < BLOCK / 2) {
        for(int offset = BLOCK / 2; offset > 0; offset /= 2) {
            if (p < offset) {
                part[p] += part[p + offset];
            }
            __syncthreads();
        }
    }
    
    if (p == 0) {
        atomicAdd(&scale_updates[filter], part[0]);
    }
}