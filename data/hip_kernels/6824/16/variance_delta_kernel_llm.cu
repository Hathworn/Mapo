#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Use shared memory for faster access to frequently used data
    extern __shared__ float shared_mean_variance[];
    float *shared_mean = shared_mean_variance;
    float *shared_variance = shared_mean_variance + filters;

    int filterIdx = threadIdx.x;
    if (filterIdx < filters) {
        shared_mean[filterIdx] = mean[filterIdx];
        shared_variance[filterIdx] = variance[filterIdx];
    }
    __syncthreads();

    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float temp_variance_delta = 0;
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            temp_variance_delta += delta[index] * (x[index] - shared_mean[i]);
        }
    }

    variance_delta[i] = temp_variance_delta * -.5f * powf(shared_variance[i] + .00001f, (float)(-3.f / 2.f));
}