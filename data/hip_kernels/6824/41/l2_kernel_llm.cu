#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop to ensure all elements are processed
    for (; i < n; i += blockDim.x * gridDim.x) {
        float diff = truth[i] - pred[i];
        error[i] = diff * diff;
        delta[i] = diff;
    }
}