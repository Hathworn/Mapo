#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Unroll loop to improve performance
    int spatial_iterations = spatial / threads;
    int remainder = spatial % threads;
    
    int j, i;
    for(j = 0; j < batch; ++j){
        for(i = 0; i < spatial_iterations * threads; i += threads){
            int index = j * spatial * filters + filter * spatial + i + id;
            local[id] += delta[index] * (x[index] - mean[filter]);
        }
        // Handle remainder spatial elements
        if (id < remainder) {
            int index = j * spatial * filters + filter * spatial + spatial_iterations * threads + id;
            local[id] += delta[index] * (x[index] - mean[filter]);
        }
    }

    __syncthreads();

    // Use atomicAdd for better parallel reduction
    if (id == 0) {
        variance_delta[filter] = 0;
    }
    atomicAdd(&variance_delta[filter], local[id]);
    __syncthreads();

    if(id == 0){
        variance_delta[filter] *= -.5f * powf(variance[filter] + .00001f, (float)(-3.f/2.f));
    }
}