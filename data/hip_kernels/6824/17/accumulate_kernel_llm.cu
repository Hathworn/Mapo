#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    // Calculate the global index for the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Return if the index is out of bounds
    if (i >= groups) return;

    // Initialize sum for the ith group
    float local_sum = 0.0f;

    // Use shared memory for partial reduction if necessary
    extern __shared__ float shared[];

    // Iterate over n elements and accumulate the sum
    for (int k = 0; k < n; ++k)
    {
        local_sum += x[k * groups + i];
    }

    // Store the local sum in the output array
    sum[i] = local_sum;
}