#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0;

    // Iterate over batch and spatial dimensions
    for(int j = 0; j < batch; ++j) {
        for(int i = id; i < spatial; i += threads) { // Stride by threads across spatial dimension
            int index = j * spatial * filters + filter * spatial + i;

            if(i < spatial) { // Ensure index is within bounds
                local[id] += powf((x[index] - mean[filter]), 2);
            }
        }
    }

    // Ensure all threads have written their values to local memory
    __syncthreads();

    // Reduce sum in shared memory by the first thread
    if(id == 0) {
        float sum = 0;
        for(int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        variance[filter] = sum / (spatial * batch - 1);
    }
}