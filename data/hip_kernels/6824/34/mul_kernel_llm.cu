#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified block index calculation
    int stride = blockDim.x * gridDim.x; // Calculate stride for increments across threads
    for (; i < N; i += stride) { // Use loop to cover all elements
        Y[i * INCY] *= X[i * INCX];
    }
}