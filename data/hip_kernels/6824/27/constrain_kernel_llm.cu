#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform operation only if index is within N
    if (i < N) {
        // Apply clamp using fminf and fmaxf to limit X[i*INCX] within [-ALPHA, ALPHA]
        float xi = X[i * INCX];
        X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, xi));
    }
}