#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;
    
    // Use shared memory for biases
    __shared__ float shared_biases[1024];
    if (threadIdx.x == 0) {
        shared_biases[filter] = biases[filter];
    }
    __syncthreads();
    
    if (offset < size) {
        output[(batch * n + filter) * size + offset] *= shared_biases[filter];
    }
}