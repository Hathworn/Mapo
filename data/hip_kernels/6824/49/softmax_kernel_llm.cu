#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;

    // Finding the largest element in the input using parallel reduction
    for (i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = (val > largest) ? val : largest;
    }

    for (i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp);
        sum += e;
        output[i * stride] = e;
    }

    // Normalize the outputs
    for (i = 0; i < n; ++i) {
        output[i * stride] /= sum;
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride_grid = blockDim.x * gridDim.x;
    
    // Using grid-stride loop for kernel launch optimization
    for (int tid = id; tid < batch * groups; tid += stride_grid) {
        int b = tid / groups;
        int g = tid % groups;
        softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
    }
}