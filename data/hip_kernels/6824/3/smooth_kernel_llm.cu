#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    // Compute unique global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    // Derive indices from global thread index
    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    int out_index = j + w * (i + h * (k + c * b));
    float delta_value = 0.0f; // Accumulate delta

    for(int l = 0; l < size; ++l){
        int cur_h = (i - size / 2) + l;
        if (cur_h >= 0 && cur_h < h) { // Check height bounds
            for(int m = 0; m < size; ++m){
                int cur_w = (j - size / 2) + m;
                if (cur_w >= 0 && cur_w < w) { // Check width bounds
                    int index = cur_w + w * (cur_h + h * (k + b * c));
                    delta_value += rate * (x[index] - x[out_index]);
                }
            }
        }
    }
    
    delta[out_index] = delta_value; // Store result
}