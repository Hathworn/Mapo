#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation
    if (i >= N) return;  // Early exit if out of bounds

    // Precompute reusable values
    int spatial_layers = spatial * layers;
    
    int in_s = i % spatial;
    int in_c = (i / spatial) % layers;
    int b = i / spatial_layers;

    int i1 = b * spatial_layers + in_c * spatial + in_s;
    int i2 = b * spatial_layers + in_s * layers + in_c;

    if (forward) out[i2] = x[i1];
    else out[i1] = x[i2];
}