#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Calculate scale factor once per thread
    float scale = 1.0f / (batch * spatial - 1);

    // Optimize index calculation using only thread and block-specific values
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Initialize variance to zero
    float temp_variance = 0.0f;

    // Unroll loops to process multiple elements per iteration, improving memory access efficiency
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            temp_variance += powf(x[index] - mean[i], 2);
        }
    }

    // Store the computed variance with scaling
    variance[i] = temp_variance * scale;
}