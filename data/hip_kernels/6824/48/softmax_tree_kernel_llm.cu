#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    // Loop unrolling for performance
    float sum = 0;
    float largest = -INFINITY;
    for (int i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = (val > largest) ? val : largest;
    }
    for (int i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp); // Combine operations
        sum += e;
        output[i * stride] = e;
    }
    float inv_sum = 1.0f / sum; // Precompute reciprocal for division
    for (int i = 0; i < n; ++i) {
        output[i * stride] *= inv_sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x; // Simplify ID calculation
    if (id >= spatial * batch * groups) return;
    
    int s = id % spatial;
    int g = (id / spatial) % groups; 
    int b = id / (spatial * groups); // Correct division order
    
    int goff = group_offset[g] * spatial;
    int boff = b * stride;
    
    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}