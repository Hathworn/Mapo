#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate linear index for the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop to allow each thread to handle multiple elements
    for (; i < N; i += blockDim.x * gridDim.x) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}