#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized mean_kernel function
__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for storing intermediate means
    __shared__ float shared_mean[1024]; // Assuming blockDim.x <= 1024
    shared_mean[threadIdx.x] = 0.0f;

    if (i < filters) {
        int j, k;
        for (j = 0; j < batch; ++j) {
            for (k = 0; k < spatial; ++k) {
                int index = j * filters * spatial + i * spatial + k;
                shared_mean[threadIdx.x] += x[index];
            }
        }
    }
    
    __syncthreads(); // Ensure all threads have updated shared memory

    // Aggregate results from shared memory
    if (i < filters) {
        mean[i] = shared_mean[threadIdx.x] * scale;
    }
}