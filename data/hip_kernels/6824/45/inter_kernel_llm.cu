#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use strided loop to allow more threads to efficiently process the data
    for (int index = i; index < (NX + NY) * B; index += gridDim.x * blockDim.x) {
        int b = index / (NX + NY);
        int j = index % (NX + NY);
        if (j < NX) {
            OUT[index] = X[b * NX + j];
        } else {
            OUT[index] = Y[b * NY + j - NX];
        }
    }
}