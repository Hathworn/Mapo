#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel function
__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation
    int stride = gridDim.x * blockDim.x;            // Calculate stride for grid

    for (int idx = i; idx < N; idx += stride) {     // Loop over entire data with stride
        X[idx * INCX] = ALPHA;
    }
}