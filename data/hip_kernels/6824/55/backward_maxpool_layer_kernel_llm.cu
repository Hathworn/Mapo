#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + 2*pad) / stride;
    int w = (in_w + 2*pad) / stride;
    int c = in_c;
    int area = (size - 1) / stride;

    // Calculate index using blockIdx and threadIdx
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Precompute necessary offsets
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    int w_offset = -pad;
    int h_offset = -pad;
    float d = 0.0f;

    // Optimize loops by unrolling and reducing unnecessary checks
    for (int l = -area; l <= area; ++l) {
        int out_h = (i - h_offset) / stride + l;
        if (out_h < 0 || out_h >= h) continue; // Skip invalid height

        for (int m = -area; m <= area; ++m) {
            int out_w = (j - w_offset) / stride + m;
            if (out_w < 0 || out_w >= w) continue; // Skip invalid width

            int out_index = out_w + w * (out_h + h * (k + c * b));
            if (indexes[out_index] == id) {
                d += delta[out_index]; // Accumulate delta if valid
            }
        }
    }
    // Update previous delta
    prev_delta[id] += d;
}