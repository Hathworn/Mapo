#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Compute the true global thread index
    int stride = blockDim.x * gridDim.x; // Compute stride for all threads in total grid

    for (int i = idx; i < n; i += stride) // Use strided loop for better memory access pattern
    {
        if (mask[i] == mask_num) 
        {
            x[i] *= scale; // Multiply only if condition is met
        }
    }
}