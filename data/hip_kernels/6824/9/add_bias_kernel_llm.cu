#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Iterate through elements with stride to cover all
    for (int idx = index; idx < n * size * batch; idx += stride) {
        int i = idx % size;
        int temp_idx = idx / size;
        int j = temp_idx % n;
        int k = temp_idx / n;
        
        output[(k * n + j) * size + i] += biases[j];
    }
}