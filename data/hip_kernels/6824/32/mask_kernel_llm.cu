#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Process only valid indices
    if (i < n && mask[i] == mask_num) {
        x[i] = mask_num;
    }
}