#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Optimize the loop by collapsing into one loop to reduce branch divergence
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index];
        }
    }

    part[p] = sum;
    __syncthreads();

    // Use reduction pattern to accumulate shared memory
    if (BLOCK >= 1024) { if (p < 512) { part[p] += part[p + 512]; } __syncthreads(); }
    if (BLOCK >= 512) { if (p < 256) { part[p] += part[p + 256]; } __syncthreads(); }
    if (BLOCK >= 256) { if (p < 128) { part[p] += part[p + 128]; } __syncthreads(); }
    if (BLOCK >= 128) { if (p < 64) { part[p] += part[p + 64]; } __syncthreads(); }
    
    // Unrolled final warp to avoid __syncthreads(); note warp size is 32
    if (p < 32) {
        if (BLOCK >= 64) part[p] += part[p + 32];
        if (BLOCK >= 32) part[p] += part[p + 16];
        if (BLOCK >= 16) part[p] += part[p + 8];
        if (BLOCK >= 8) part[p] += part[p + 4];
        if (BLOCK >= 4) part[p] += part[p + 2];
        if (BLOCK >= 2) part[p] += part[p + 1];
    }

    if (p == 0) {
        bias_updates[filter] += part[0];
    }
}