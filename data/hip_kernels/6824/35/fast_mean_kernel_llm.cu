#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Unroll the loop to improve memory access efficiency
    for(int j = 0; j < batch; ++j){
        for(int i = id; i < spatial; i += threads){
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += x[index];
        }
    }

    __syncthreads();

    // Use parallel reduction to sum local array
    if (id < warpSize)
    {
        for (int offset = warpSize; offset < threads; offset += warpSize)
        {
            local[id] += local[id + offset];
        }
    }

    __syncthreads();

    // Final reduction and writing result by a single thread
    if (id == 0)
    {
        float sum = 0;
        for (int i = 0; i < warpSize; ++i)
        {
            sum += local[i];
        }
        mean[filter] = sum / (spatial * batch);
    }
}