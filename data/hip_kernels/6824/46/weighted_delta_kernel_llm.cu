#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop to handle cases where threads exceed `n`
    for (int stride = i; stride < n; stride += blockDim.x * gridDim.x) {
        if(da) da[stride] += dc[stride] * s[stride];  // Update `da` if not null
        if(db) db[stride] += dc[stride] * (1 - s[stride]);  // Update `db` if not null
        ds[stride] += dc[stride] * (a[stride] - b[stride]);  // Always update `ds`
    }
}