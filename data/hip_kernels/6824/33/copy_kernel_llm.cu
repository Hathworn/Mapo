#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate global thread index using a 1D grid of 1D blocks
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread index is within bounds before copying data
    if (i < N)
    {
        Y[i*INCY + OFFY] = X[i*INCX + OFFX];
    }
}