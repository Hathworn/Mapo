#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure accessing within bounds
    if(i < n) {
        // Perform multiply-add operation
        c[i] += a[i] * b[i];
    }
}