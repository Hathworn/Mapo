#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index < N) {  // Simplify condition check to reduce divergent branches
        int f = (index / spatial) % filters;
        float var_sqrt = rsqrtf(variance[f] + 1e-5f);  // Use rsqrtf for faster computation
        float scale = 2.0f / (spatial * batch);  // Precompute repeated expression

        // Optimize calculation by using precomputed values
        delta[index] = delta[index] * var_sqrt 
                     + variance_delta[f] * scale * (x[index] - mean[f]) 
                     + mean_delta[f] / (spatial * batch);
    }
}