#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    // Calculate global thread index
    int s = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    
    // Exit if the thread is out of bounds
    if (s >= size) return;

    int i = 0;
    float mean = 0.0f;
    
    // Optimized: Using shared memory for faster access
    __shared__ float shared_mean;

    // Compute mean using atomic addition for shared memory access
    for (i = 0; i < n; ++i) {
        atomicAdd(&shared_mean, fabsf(input[i * size + s]));
    }
    __syncthreads();
    
    // Final mean calculation outside the loop
    mean = shared_mean / n;
    
    // Write the binarized values using the computed mean
    for (i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}