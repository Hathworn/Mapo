#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread ID more efficiently using grid-stride loop
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    for (; i < N; i += stride) {
        float value = X[i * INCX];  // Access the array with offset
        if ((value * value) < (ALPHA * ALPHA)) {
            X[i * INCX] = 0;  // Set to 0 if condition is met
        }
    }
}