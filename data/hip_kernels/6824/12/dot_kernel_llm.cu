#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Optimize by computing 3D thread index based on batch, n, and size
    int batch_index = blockIdx.z;
    int row_index = blockIdx.y;
    int col_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (row_index >= n || col_index >= n || col_index <= row_index) return;

    float sum = 0.0;
    float norm1 = 0.0;
    float norm2 = 0.0;

    // Cache indices for memory access optimization
    int row_base = batch_index * size * n + row_index * size;
    int col_base = batch_index * size * n + col_index * size;

    // Use shared memory for partial results
    __shared__ float shared_sum[1024];
    __shared__ float shared_norm1[1024];
    __shared__ float shared_norm2[1024];

    shared_sum[threadIdx.x] = 0.0;
    shared_norm1[threadIdx.x] = 0.0;
    shared_norm2[threadIdx.x] = 0.0;

    // Parallelize the dot product calculation
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        int i1 = row_base + i;
        int i2 = col_base + i;
        float val1 = output[i1];
        float val2 = output[i2];
        shared_sum[threadIdx.x] += val1 * val2;
        shared_norm1[threadIdx.x] += val1 * val1;
        shared_norm2[threadIdx.x] += val2 * val2;
    }

    // Synchronize threads before reduction
    __syncthreads();

    // Reduce shared memory results
    if (threadIdx.x == 0) {
        for (int i = 1; i < blockDim.x; i++) {
            shared_sum[0] += shared_sum[i];
            shared_norm1[0] += shared_norm1[i];
            shared_norm2[0] += shared_norm2[i];
        }

        float norm1 = sqrt(shared_norm1[0]);
        float norm2 = sqrt(shared_norm2[0]);
        float norm = norm1 * norm2;
        shared_sum[0] = shared_sum[0] / norm;

        // Update delta arrays
        for (int i = 0; i < size; ++i) {
            int i1 = row_base + i;
            int i2 = col_base + i;
            atomicAdd(&delta[i1], -scale * shared_sum[0] * output[i2] / norm);
            atomicAdd(&delta[i2], -scale * shared_sum[0] * output[i1] / norm);
        }
    }
}