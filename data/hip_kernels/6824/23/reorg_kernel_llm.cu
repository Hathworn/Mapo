#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Precompute values that are used multiple times
    int width_stride = w * stride;
    int area_stride = width_stride * h * stride;
    int out_c_total = c / (stride * stride);

    int in_w = i % w;
    i /= w;
    int in_h = i % h;
    i /= h;
    int in_c = i % c;
    i /= c;
    int b = i % batch;

    int c2 = in_c % out_c_total;
    int offset = in_c / out_c_total;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;

    int out_index = w2 + width_stride * (h2 + h * stride * (c2 + out_c_total * b));

    // Use ternary operator for compactness
    if (forward) 
        out[out_index] = x[in_index];
    else 
        out[in_index] = x[out_index];
}