#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize block index calculation

    if(i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);

        // Use ternary operator for compact control flow
        error[i] = (abs_val < 1) ? (diff * diff) : (2 * abs_val - 1);
        delta[i] = (abs_val < 1) ? diff : copysignf(1.0f, diff); // Use copysignf for sign
    }
}