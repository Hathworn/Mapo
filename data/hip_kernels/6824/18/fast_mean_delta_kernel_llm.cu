#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0;

    // Unrolling loops for better performance
    for (int j = 0; j < batch; ++j) {
        int base_idx = j * spatial * filters + filter * spatial;
        for (int i = id; i < spatial; i += threads) {
            local[id] += delta[base_idx + i];
        }
    }

    __syncthreads();

    // Reduce sum using parallel reduction
    for (int stride = threads / 2; stride > 0; stride /= 2) {
        if (id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    if (id == 0) {
        mean_delta[filter] = local[0] * (-1.f / sqrtf(variance[filter] + .00001f));
    }
}