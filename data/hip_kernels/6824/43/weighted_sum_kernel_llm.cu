#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate the index using block and thread values
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate the stride for grid-stride loop

    // Use grid-stride loop to allow the kernel to handle any array size
    for (int idx = i; idx < n; idx += stride) {
        c[idx] = s[idx] * a[idx] + (1 - s[idx]) * (b ? b[idx] : 0);
    }
}