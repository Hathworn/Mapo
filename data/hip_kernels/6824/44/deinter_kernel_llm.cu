#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Use 1D grid for computing i
    int totalSize = (NX + NY) * B;
    int stride = gridDim.x * blockDim.x;  // Calculate stride for grid-stride loop

    for (; i < totalSize; i += stride) {  // Grid-stride loop for better utilization
        int b = i / (NX + NY);
        int j = i % (NX + NY);

        if (j < NX) {
            if (X) X[b * NX + j] += OUT[i];  // Simplified conditional operation
        } else {
            if (Y) Y[b * NY + j - NX] += OUT[i];  // Simplified conditional operation
        }
    }
}