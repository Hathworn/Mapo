#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory to enhance memory access patterns if needed
// Use flexible grid-stride loop for better workload distribution 
__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = idx; i < N; i += stride)
    {
        X[i * INCX] += ALPHA;
    }
}