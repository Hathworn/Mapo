#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized adam_kernel with thread-coarsening to improve memory access patterns
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int stride = gridDim.x * gridDim.y * blockDim.x;
    
    float rate_sqrt_B2 = rate * sqrtf(1.f - powf(B2, t)) / (1.f - powf(B1, t));
    
    for (int i = index; i < N; i += stride) {
        x[i] += rate_sqrt_B2 * m[i] / (sqrtf(v[i]) + eps);
    }
}