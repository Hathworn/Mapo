#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    // Compute the global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n * c) return;  // Overlapping kernel launch bounds check

    // Compute batch and channel indices
    int k = id % c;  // Channel index
    int b = id / c;  // Batch index

    // Precompute repeated expressions
    int in_offset = k + c * b;
    int out_index = in_offset;
    float val = out_delta[out_index] / (w * h);

    // Iterate over width and height
    for (int i = 0; i < w * h; ++i) {
        int in_index = i + w * h * in_offset;  // Calculate input index
        in_delta[in_index] += val;  // Accumulate result
    }
}