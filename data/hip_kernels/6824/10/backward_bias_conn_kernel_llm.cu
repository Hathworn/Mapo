#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation for 1D grid
    if (index >= n) return;

    float sum = 0;
    for (int b = 0; b < batch; ++b) {
        sum += delta[b * n + index]; // Accumulate delta values
    }

    atomicAdd(&bias_updates[index], sum); // Use atomicAdd for thread safety
}