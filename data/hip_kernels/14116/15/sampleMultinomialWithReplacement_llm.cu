#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;
    while (end - start > 0) {
        int mid = start + (end - start) / 2;
        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }
    if (start == size) {
        start = 0;
    }
    return start;
}

__global__ void sampleMultinomialWithReplacement(hiprandStateMtgp32* state, int totalSamples, float* dest, long distributions, int categories, float* normDistPrefixSum) {
    // Use blockIdx.y for additional parallelism if needed
    long distPerBlock = blockDim.y * gridDim.x;
    
    // Distribute work among blocks to reduce iteration overhead
    for (long curDist = blockIdx.x + blockIdx.y * gridDim.x;
         curDist < distributions;
         curDist += distPerBlock) {

        // Optimize loop increments and use parallelism within a block
        for (int sampleBase = threadIdx.y;
             sampleBase < totalSamples;
             sampleBase += blockDim.y) {
            
            float r = hiprand_uniform(&state[blockIdx.x]);
            
            int choice = binarySearchForMultinomial(
                normDistPrefixSum + curDist * categories,
                categories, r);
                
            dest[curDist * totalSamples + sampleBase] = static_cast<float>(choice) + 1.0f;
        }
    }
}