#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getValue(float *outdata, float *indata) {
    // Optimize by directly assigning the value based on the condition
    float result = (indata == nullptr) ? 3.0f : 2.0f;
    outdata[0] = result;
}