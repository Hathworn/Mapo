#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void floattoint(int *out, float *in) {
    // Use thread index to support parallel execution
    int idx = threadIdx.x;
    // Compute conversion within bounds check for efficiency
    if (idx < 1) {
        out[idx] = __float_as_int(in[idx]);
    }
}