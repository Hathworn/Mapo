#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getValue(float4 *outdata, float *indata) {
    // Use thread index for parallel execution
    int idx = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;

    // Load data efficiently using the thread index
    if (idx == 0) {
        outdata[idx] = make_float4(indata[0], indata[3], indata[1], indata[2]);
    }
}