#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getValueChar(char *outdata, char *indata) {
    // Use blockIdx and threadIdx to handle data more efficiently
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        outdata[idx] = indata[idx] + 3;
    }
}