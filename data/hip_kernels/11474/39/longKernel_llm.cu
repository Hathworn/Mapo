#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void longKernel(float *data, int N, float value) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Check bounds and perform operation
    if (tid < N) {
        data[tid] += value;
    }
}