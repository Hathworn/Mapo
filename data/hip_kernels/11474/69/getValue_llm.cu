#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getValue(float *outdata, float *indata) {
    // Use aliasing for better readability
    float input = indata[0];
    
    // Perform computation
    float result = input + 3.0f;
    
    // Store result
    outdata[0] = result;
}