#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void main_set(float *data, float val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Use block index for better scalability
    data[idx] = val;  // Assign value to correct index in array
}