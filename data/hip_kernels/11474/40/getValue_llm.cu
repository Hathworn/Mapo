#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getValue(float *outdata, const float *indata) {
    // Check if the address of indata is null
    outdata[0] = (__ldg(indata) == 0) ? 3.0f : 2.0f;  // Use __ldg for faster global memory access
}