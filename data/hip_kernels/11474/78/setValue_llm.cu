#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void setValue(float *data, int idx, float value) {
    if(blockIdx.x == 0 && threadIdx.x == 0) { // Ensure only one thread writes by checking blockIdx
        data[idx] = value;
    }
}