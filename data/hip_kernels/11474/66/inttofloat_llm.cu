#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inttofloat(float *out, int *in) {
    // Load integer value from global memory to a register
    int intValue = in[0];

    // Use bitwise operations or type casting to convert to float
    float floatValue = __int_as_float(intValue);

    // Store the result back to the global memory
    out[0] = floatValue;
}