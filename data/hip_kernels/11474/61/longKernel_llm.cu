#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void longKernel(float *data, int N, float value) {
    // Use threadIdx and blockIdx to parallelize the loop
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        data[idx] += value;  // Only process assigned element
    }
}

```
