#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mykernel(float *d1, float *d2, float *d3, float *d4, float *d5) {
    // Use threadIdx.x to vectorize operations on d* arrays
    int i = threadIdx.x;

    if (i < 5) {
        float* arrays[] = {d1, d2, d3, d4, d5};
        arrays[i][0] = 123.0f;  // Assign 123.0f to the first element of each array
    }
}