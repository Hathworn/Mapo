#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getValue(float *outdata, float *indata) {
    // Use block and thread index to allow parallel execution
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) { // Process the first element only
        outdata[idx] = indata[idx] + 3.0f;
    }
}