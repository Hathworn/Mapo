#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void incrValue(float *data, int idx, float value) {
    // Single thread in the grid performs the increment operation for efficiency
    if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
        data[idx] += value;
    }
}