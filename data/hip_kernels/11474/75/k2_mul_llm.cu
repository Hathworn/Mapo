#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel to handle larger workloads
__global__ void k2_mul(float *data, float val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (idx < N) { // Use a boundary check (assuming N is defined elsewhere as data size)
        data[idx] *= val;
    }
}