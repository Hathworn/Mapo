#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setValue(float *data, int idx, float value) {
    // Use block-based indexing to assign value
    if(blockIdx.x == 0 && threadIdx.x == 0) {
        data[idx] = value;
    }
}