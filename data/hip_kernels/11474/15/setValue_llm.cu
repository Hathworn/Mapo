#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

__global__ void setValue(float *data, int idx, float value) {
    // Use a single thread block and assign work to shared memory
    __shared__ float sharedValue;

    if (threadIdx.x == 0) {
        sharedValue = value;  // Assign value to shared memory
    }
    __syncthreads();  // Synchronize threads within block

    if (threadIdx.x == 0) {
        data[idx] = sharedValue;  // Write to global memory
    }
}