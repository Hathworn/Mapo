#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize setValue kernel function to avoid warp divergence
__global__ void setValue(float *data, int idx, float value) {
    // Use a single thread to set the value, avoiding unnecessary conditional
    if(blockIdx.x == 0 && threadIdx.x == 0) {
        data[idx] = value;
    }
}