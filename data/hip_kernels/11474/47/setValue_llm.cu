#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setValue(float *data, int idx, float value) {
    // Only one thread in the block needs to update the data
    if(blockIdx.x == 0 && threadIdx.x == 0) {
        data[idx] = value;
    }
}