#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void wipe(int *buffer, int length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Global index for threads
    int stride = blockDim.x * gridDim.x; // Grid-stride loop increment
    length >>= 5;

    for(int i = index; i < length; i += stride) {
        buffer[(i << 5) + threadIdx.x] = -1; // Set buffer value to -1
    }
}