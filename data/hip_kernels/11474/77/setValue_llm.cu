#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use blockIdx.x and blockDim.x to determine a unique thread
__global__ void setValue(float *data, int idx, float value) {
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique global thread index

    // Ensure only a single thread performs the write operation to prevent race conditions
    if(globalIdx == 0) {
        data[idx] = value;  // Write the value at the specified index
    }
}