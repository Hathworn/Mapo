#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void longKernel(float *data, int N, float value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (idx < N) { // Ensure within bounds
        data[idx] += value; // Perform operation
    }
}