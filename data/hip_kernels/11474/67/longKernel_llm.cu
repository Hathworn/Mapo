#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void longKernel(float *data, int N, float value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global index
    int stride = gridDim.x * blockDim.x;  // Calculate stride

    for (int i = idx; i < N; i += stride) {
        data[i] += value;  // Update element at index i
    }
}