#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void k1_incr(float *data, float val) {
    // Calculate global thread index for 1D grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    // Safeguard against accessing memory out of bounds
    if (idx < N) { // Assume N is the size of data defined elsewhere
        data[idx] += val;
    }
}