#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mykernel(int *int1, float *f1, int *int2) {
    // Use shared memory to optimize data movement
    __shared__ int sharedInt;
    __shared__ float sharedFloat;

    if (threadIdx.x == 0) {
        sharedInt = int1[0];
        sharedFloat = *reinterpret_cast<float*>(&sharedInt);
        f1[0] = sharedFloat;
        int2[0] = *reinterpret_cast<int*>(&sharedFloat);
    }
}