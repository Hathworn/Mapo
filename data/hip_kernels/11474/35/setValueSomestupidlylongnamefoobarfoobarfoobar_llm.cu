#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void setValueSomestupidlylongnamefoobarfoobarfoobarhaha(float *data, int idx, float value) {
    data[idx] = value;
}

__device__ float bar(float a, float b) {
    return a + b;
}

__global__ void setValueSomestupidlylongnamefoobarfoobarfoobar(float *data, int idx, float value) {
    // Use blockIdx.x and blockDim.x to allow parallel threads to set values
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalIdx == idx) {
        setValueSomestupidlylongnamefoobarfoobarfoobarhaha(data, idx, value);
    }
}