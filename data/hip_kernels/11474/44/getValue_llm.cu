#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getValue(float *outdata, float *indata) {
    // Use ternary operator with conditional memory operations
    outdata[0] = (*indata == 0.0f) ? 3.0f : 2.0f;
}