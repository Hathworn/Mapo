#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diag_normalize(double *A, double *I, int n, int i) {
    // Compute the unique thread index within the grid
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Only one thread needs to perform the operation
    if (tid == i && tid < n) { 
        double diag = A[i * n + i];
        if (diag != 0) {  // Ensure we're not dividing by zero
            I[tid * n + tid] /= diag;
            A[tid * n + tid] /= diag;
        }
    }
}