#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nodiag_normalize(double *A, double *I, int n, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure only relevant threads execute the division
    if (x == i && x != y && y < n) {
        double diag_val = A[i * n + i];  // Cache diagonal element
        I[x * n + y] /= diag_val;
        A[x * n + y] /= diag_val;
    }
}