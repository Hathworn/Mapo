#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function to set specific elements to zero
__global__ void set_zero(double *A, int n, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure indices are within bounds and simplify conditional logic
    if (x < n && y == i && x != i) {
        A[x * n + y] = 0;
    }
}