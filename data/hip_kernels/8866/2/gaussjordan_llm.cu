#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaussjordan(double *A, double *I, int n, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Create a shared memory space to store row i of A and I
    __shared__ double shared_Ai[1024];
    __shared__ double shared_Ii[1024];
    
    // Load row i into shared memory
    if (y < n) {
        shared_Ai[y] = A[i * n + y];
        shared_Ii[y] = I[i * n + y];
    }
    __syncthreads();

    if (x < n && y < n) {
        if (x != i) {
            I[x * n + y] -= shared_Ii[y] * A[x * n + i];
            if (y != i) {
                A[x * n + y] -= shared_Ai[y] * A[x * n + i];
            }
        }
    }
}