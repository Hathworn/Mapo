#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Ensure within bounds
    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Compute target coordinates and offsets
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py;
    float dx = modff (cx, &px);
    float dy = modff (cy, &py);

    // integer coords for target pixel
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);
    float value = src[image_row_offset + j];
    float weight;
    
    // Function to conditionally add weight and value
    auto conditionalAdd = [&](int x, int y, float w) {
        if (!(x >= w || x < 0 || y >= h || y < 0)) {
            _atomicAdd(dst + y * image_stride + x, value * w);
            _atomicAdd(normalization_factor + y * image_stride + x, w);
        }
    };
    
    // Fill pixels with calculated weights
    weight = dx * dy;
    conditionalAdd(tx, ty, weight);

    tx -= 1;
    weight = (1.0f - dx) * dy;
    conditionalAdd(tx, ty, weight);

    ty -= 1;
    weight = (1.0f - dx) * (1.0f - dy);
    conditionalAdd(tx, ty, weight);

    tx += 1;
    weight = dx * (1.0f - dy);
    conditionalAdd(tx, ty, weight);
}