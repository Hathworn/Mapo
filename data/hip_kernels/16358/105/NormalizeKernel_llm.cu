#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global thread position once
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop to cover all elements
    for (; i < h; i += gridDim.y * blockDim.y) {
        for (int jj = j; jj < w; jj += gridDim.x * blockDim.x) {
            const int pos = i * s + jj;

            float scale = normalization_factor[pos];

            float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

            image[pos] *= invScale;
        }
    }
}