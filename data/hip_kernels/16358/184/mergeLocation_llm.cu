#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];
        
        // Minimize global memory access by using shared memory for frequently accessed data.
        __shared__ float sharedX[256];
        __shared__ float sharedY[256];
        
        sharedX[threadIdx.x] = loc.x * scale;
        sharedY[threadIdx.x] = loc.y * scale;
        
        __syncthreads(); // Ensure all threads have written to shared memory.
        
        x[ptidx] = sharedX[threadIdx.x];
        y[ptidx] = sharedY[threadIdx.x];
    }
}