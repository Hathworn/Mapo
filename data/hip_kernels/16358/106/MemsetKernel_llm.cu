#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global thread ID
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds and assign value
    if (j < w && i < h)
    {
        int pos = i * w + j;
        image[pos] = value;
    }
}