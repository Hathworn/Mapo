#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Using a loop to handle out-of-bounds checking and reducing the number of kernel launches when len is large
    for (int i = pos; i < len; i += blockDim.x * gridDim.x) {
        d_res[i] = d_src[i] * scale;
    }
}