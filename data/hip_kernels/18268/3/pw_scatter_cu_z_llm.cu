#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_scatter_cu_z(double *c, const double *pwcc, const double scale, const int ngpts, const int nmaps, const int *ghatmap) {

    const int igpt = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x; // Calculate global thread index

    if (igpt < ngpts) {
        int idx = 2 * igpt; // Precompute index
        int mapIdx = 2 * ghatmap[igpt]; // Precompute map index

        // Perform scattering operation
        double valueReal = scale * pwcc[idx];
        double valueImag = scale * pwcc[idx + 1];

        c[mapIdx] = valueReal;
        c[mapIdx + 1] = valueImag;

        // Check and process for nmaps
        if (nmaps == 2) {
            int mapIdx2 = 2 * ghatmap[igpt + ngpts];
            c[mapIdx2] = valueReal;
            c[mapIdx2 + 1] = -valueImag;
        }
    }
}