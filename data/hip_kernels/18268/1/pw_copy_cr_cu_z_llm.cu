#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_copy_cr_cu_z(const double *zin, double *dout, const int n) {
    // Use shared memory for better memory coalescing
    extern __shared__ double sharedMem[];

    const int igpt = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;

    if (igpt < n) {
        // Load data into shared memory
        sharedMem[threadIdx.x] = zin[2 * igpt];
        __syncthreads();
        
        // Write back to global memory
        dout[igpt] = sharedMem[threadIdx.x];
    }
}