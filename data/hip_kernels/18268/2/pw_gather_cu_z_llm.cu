#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_gather_cu_z(double *pwcc, const double *c, const double scale, const int ngpts, const int *ghatmap) {
    // Use a 1D block and grid for better performance on linear access
    int igpt = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize memory access by reducing the number of multiplications
    if (igpt < ngpts) {
        int c_index = 2 * ghatmap[igpt];
        pwcc[2 * igpt] = scale * c[c_index];
        pwcc[2 * igpt + 1] = scale * c[c_index + 1];
    }
}