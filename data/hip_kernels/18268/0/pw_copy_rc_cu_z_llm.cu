#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_copy_rc_cu_z(const double *din, double *zout, const int n) {
    const int igpt = blockDim.x * (gridDim.x * blockIdx.y + blockIdx.x) + threadIdx.x;

    // Check and directly store the index calculation
    if (igpt < n) {
        int idx = 2 * igpt; // Pre-calculate index to optimize access
        zout[idx] = din[igpt];
        zout[idx + 1] = 0.0e0;
    }
}