#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PadMatrixInLeadingDimensionKernel(const int8_t* src, int8_t* dst, int col_src, int col_dst) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < col_src) {  // Avoid redundant condition checks inside loop
        *(dst + blockIdx.x * col_dst + idx) = *(src + blockIdx.x * col_src + idx);
    }
}