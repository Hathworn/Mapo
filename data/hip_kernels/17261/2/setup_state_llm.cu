#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Optimize by initializing per thread state
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}