#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Use a single global memory access per output element
    int32_t idx = blockIdx.x * N + threadIdx.x;  // Calculate global index once
    for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
        output[idx + i] = -row_sum[blockIdx.x];  // Use updated index
    }
}