#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Use shared memory for storing batch and class indices to avoid redundant writes
    __shared__ int64_t shared_batch_index;
    __shared__ int64_t shared_class_index;
    
    if(threadIdx.x == 0) {
        shared_batch_index = batch_index;
        shared_class_index = class_index;
    }
    __syncthreads();
    
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int idx = global_idx; idx < num_elements; idx += blockDim.x * gridDim.x) {
        to_normalize[idx * 3] = shared_batch_index;
        to_normalize[idx * 3 + 1] = shared_class_index;
        to_normalize[idx * 3 + 2] = static_cast<int64_t>(original[idx]);
    }
}