#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixAB(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, int32_t K_A_B, int32_t N) {
    // Use blockIdx.x and threadIdx.x to calculate unique thread index
    int32_t index = blockIdx.x * blockDim.x + threadIdx.x; 

    // Ensure that the thread index is within bounds
    if (index < N) {  
        output[blockIdx.x * N + index] = K_A_B - row_sum[blockIdx.x] - col_sum[index];
    }
}
```
