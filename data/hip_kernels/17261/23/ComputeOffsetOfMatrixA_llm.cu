#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    int32_t idx = blockIdx.x * N + threadIdx.x;
    // Using grid-stride loop to handle larger N
    for (int32_t i = idx; i < (blockIdx.x + 1) * N; i += blockDim.x) {
        output[i] = -col_sum[i % N];
    }
}