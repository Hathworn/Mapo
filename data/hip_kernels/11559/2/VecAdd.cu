#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(float* A, float *B, float *C)
{
int idx = threadIdx.x;
C[idx] = A[idx] + B[idx];
}