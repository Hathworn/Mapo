#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAdd(float* A, float *B, float *C, int n)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check bounds to avoid accessing out-of-range elements
    if (idx < n) {
        C[idx] = A[idx] + B[idx];
    }
}