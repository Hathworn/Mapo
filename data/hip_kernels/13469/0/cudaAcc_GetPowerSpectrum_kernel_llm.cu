#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define B 2

// Kernel function to calculate power spectrum
__global__ void cudaAcc_GetPowerSpectrum_kernel(int NumDataPoints, float2* FreqData, float* PowerSpectrum) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundary to prevent accessing out-of-bounds memory
    if (i < NumDataPoints) {
        float ax = FreqData[i].x;
        float ay = FreqData[i].y;
        // Use optimized fused multiply-add operations
        PowerSpectrum[i] = __fmaf_rn(ax, ax, __fmul_rn(ay, ay));
    }
}