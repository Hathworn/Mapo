#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define B 2

__global__ void cudaAcc_GetPowerSpectrum_kernel2(int NumDataPoints, float2* FreqData, float* PowerSpectrum)
{
    // Calculate starting index for the thread
    const int index = blockIdx.x * blockDim.x * B + threadIdx.x;

    // Pre-loading data into registers for coalesced memory access
    float2 freqData[B];

    #pragma unroll
    for (int k = 0; k < B; k++)
    {
        freqData[k] = FreqData[index + k * blockDim.x];
    }

    #pragma unroll
    for (int k = 0; k < B; k++)
    {
        float ax = freqData[k].x;
        float ay = freqData[k].y;
        PowerSpectrum[index + k * blockDim.x] = __fadd_rn(__fmul_rn(ax, ax), __fmul_rn(ay, ay));
    }
}