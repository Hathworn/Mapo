#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0;

    // First reduction level: compute partial dot products
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Second reduction level: accumulate total dot product using atomic addition for efficiency
    if (idx == 0) {
        tmp = 0.0f;
        for (int i = 0; i < blockDim.x; ++i) {
            tmp += reduction_buffer[i];
        }
        reduction_buffer[0] = tmp;
    }
    __syncthreads();

    // Compute gradients
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}