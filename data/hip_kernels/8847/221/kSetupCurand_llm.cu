#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate global thread index once
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    // Initialize hiprand state with pre-calculated index
    hiprand_init(seed, tidx, 0, &state[tidx]);
}