#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    
    // Use shared memory for partial results to improve memory access efficiency
    extern __shared__ float sharedSrc[];

    // Cooperative loading of source into shared memory
    for (uint i = threadIdx.x; i < srcWidth * srcHeight; i += blockDim.x) {
        sharedSrc[i] = src[i];
    }
    __syncthreads();  // Ensure all thread loads into shared memory are complete
    
    // Iterate over the output elements
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        tgt[i] = sharedSrc[srcY * srcWidth + srcX];
    }
}