#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        // Calculate log probability of the label
        labelLogProbs[tx] = __logf(labelp);

        // If label probability is not equal to max probability, set correctProbs to 0
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            // Count number of labels with probability equal to max
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp) ? 1 : 0;
            }
            // Calculate correct probability as reciprocal of numMax
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}