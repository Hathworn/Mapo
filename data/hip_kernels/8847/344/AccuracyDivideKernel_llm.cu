#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use block-level parallelism, assuming one element to compute
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}