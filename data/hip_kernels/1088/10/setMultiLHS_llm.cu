#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setMultiLHS(double* dsMulti, double* dlMulti, double* diagMulti, double* duMulti, double* dwMulti, double a, double b, double c, double d, double e, int nx, int batchCount)
{
    // Calculate thread's global index
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIdy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds for both indices
    if (globalIdx < batchCount && globalIdy < nx)
    {
        int index = globalIdy * batchCount + globalIdx;  // Compute index inside bounds

        // Assign values to the matrices
        dsMulti[index] = a;
        dlMulti[index] = b;
        diagMulti[index] = c;
        duMulti[index] = d;
        dwMulti[index] = e;
    }
}