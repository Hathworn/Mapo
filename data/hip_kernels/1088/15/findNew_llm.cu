#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void findNew(double* cCurr, double* cBar, double* cHalf, int nx)
{
    // Calculate global thread index
    int index = blockDim.x * blockIdx.x + threadIdx.x + 
                (blockDim.y * blockIdx.y + threadIdx.y) * nx;

    // Update current matrix value
    if(index < nx * gridDim.y * blockDim.y) { // Check bounds
        cCurr[index] = cBar[index] + cHalf[index];
    }
}