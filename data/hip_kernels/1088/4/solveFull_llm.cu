#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void solveFull ( double* data, double* inv1, double* inv2, const int nx, const int nBatch )
{
    // Matrix index
    int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
    int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Check boundary
    if (globalIdy >= nx - 2) return;

    // Calculate reused data indices for performance
    int globalBatchIdx = globalIdx;
    int oldNx2Idx = (nx - 2) * nBatch + globalBatchIdx; // Two points from end
    int oldNx1Idx = (nx - 1) * nBatch + globalBatchIdx; // One point from end
    
    // Load reused data into registers
    double oldNx2 = data[oldNx2Idx];
    double oldNx1 = data[oldNx1Idx];
 
    // Set index being computed
    int index = globalIdy * nBatch + globalBatchIdx;

    // Perform computation
    data[index] -= inv1[index] * oldNx2 + inv2[index] * oldNx1;
}