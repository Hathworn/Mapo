#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void findRHS(double* cOld, double* cCurr, double* cHalf, double* cNonLinRHS, int nx)
{
    // Calculate global index directly in 1D
    int index = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * nx;

    // Optimize memory access by reducing the global memory fetch count
    double currVal = cCurr[index];
    double oldVal = cOld[index];

    // Update cHalf with optimized arithmetic operations
    cHalf[index] += - (2.0 / 3.0) * (currVal - oldVal) + cNonLinRHS[index];

    // Set cOld to current value
    cOld[index] = currVal;
}