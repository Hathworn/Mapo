#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void findRHS(double* cOld, double* cCurr, double* cHalf, double* cNonLinRHS, int nx)
{
    // Compute unique global thread index in x and y dimensions
    int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
    int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

    // Compute linear index from 2D block
    int index = globalIdy * nx + globalIdx;
    
    // Check if index is within valid range to avoid out-of-bounds access
    if (globalIdx < nx && globalIdy < nx) {
        // Optimize RHS calculation using combined operations
        double delta = (2.0 / 3.0) * (cOld[index] - cCurr[index]);
        cHalf[index] += delta + cNonLinRHS[index];

        // Directly update cOld with cCurr
        cOld[index] = cCurr[index];
    }
}
```
