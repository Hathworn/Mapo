#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel2DYp(double* dataOutput, double* dataInput, double* boundaryTop, double* boundaryBottom, const double* weights, const int numSten, const int numStenTop, const int numStenBottom, const int nxLocal, const int nyLocal, const int BLOCK_Y, const int nx, const int nyTile) {
    // Allocate shared memory
    extern __shared__ double memory[];

    double* arrayLocal = memory;
    double* weigthsLocal = &memory[nxLocal * (BLOCK_Y + numStenTop + numStenBottom)];

    // Move the weights into shared memory
    #pragma unroll
    for (int k = 0; k < numSten; ++k) {
        weigthsLocal[k] = weights[k];
    }

    // True matrix index
    int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
    int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

    // Local matrix index
    int localIdx = threadIdx.x;
    int localIdy = threadIdx.y + numStenTop;

    // Local sum variable
    double sum = 0.0;

    // Common operations in all cases
    auto processStencil = [&](double *topBoundary, double *bottomBoundary) {
        arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

        if (threadIdx.y < numStenTop) {
            arrayLocal[threadIdx.y * nxLocal + localIdx] = topBoundary[threadIdx.y * nx + globalIdx];
        }

        if (threadIdx.y < numStenBottom) {
            arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = bottomBoundary[(localIdy + BLOCK_Y) * nxLocal + localIdx];
        }

        __syncthreads();

        int stenSet = threadIdx.y * nxLocal + localIdx;

        #pragma unroll
        for (int k = 0; k < numSten; ++k) {
            sum += weigthsLocal[k] * arrayLocal[stenSet + k * nxLocal];
        }

        __syncthreads();
        dataOutput[globalIdy * nx + globalIdx] = sum;
    };

    // Set all interior blocks
    if (blockIdx.y != 0 && blockIdx.y != nyTile / BLOCK_Y - 1) {
        processStencil(dataInput - numStenTop * nx, dataInput + BLOCK_Y * nx);
    }

    // Set all top boundary blocks
    if (blockIdx.y == 0) {
        processStencil(boundaryTop, dataInput + BLOCK_Y * nx);
    }

    // Set the bottom boundary blocks
    if (blockIdx.y == nyTile / BLOCK_Y - 1) {
        processStencil(dataInput - numStenTop * nx, boundaryBottom);
    }
}
```
