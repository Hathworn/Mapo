#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void findCBar(double* cOld, double* cCurr, double* cBar, int nx)
{
    // Matrix index
    int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;

    // Avoid index calculation for unnecessary threads
    if (globalIdx >= nx) return;

    int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;
    int index = globalIdy * nx + globalIdx;

    // Ensure index is within bounds before accessing arrays
    if (index < nx * gridDim.y * blockDim.y) {
        // Compute cBar
        cBar[index] = 2.0 * cCurr[index] - cOld[index];
    }
}