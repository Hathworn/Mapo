#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void solveFull(double* data, double* inv1, double* inv2, const int nx, const int nBatch)
{
    // Matrix index
    int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
    int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

    // Guard clause to prevent accessing out-of-bounds memory
    if (globalIdx >= nBatch || globalIdy >= nx - 2) return;
    
    // Set values to last two entries in array
    double oldNx2 = data[(nx - 2) * nBatch + globalIdx]; // Two points from end
    double oldNx1 = data[(nx - 1) * nBatch + globalIdx]; // One point from end

    // Prevent redundant calculations outside the intended index range
    if (globalIdy < nx - 2)
    {
        int index = globalIdy * nBatch + globalIdx;
        data[index] -= (inv1[index] * oldNx2 + inv2[index] * oldNx1); // Optimize subtract operation
    }
}