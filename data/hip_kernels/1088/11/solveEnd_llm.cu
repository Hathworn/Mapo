#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void solveEnd ( double* data,  const double a, const double b, const double d, const double e,  const double omega_11, const double omega_12, const double omega_21, const double omega_22,  const int nx, const int nBatch )
{
    // Matrix index
    int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;

    // Bounds checking to prevent out-of-bounds access
    if (globalIdx >= nBatch * nx) return;

    // Pre-calculating repeated index calculations
    int baseIdx = globalIdx + (nx - 4) * nBatch; 

    // Last two vectors
    double newNx2;
    double newNx1;

    // Compute lambda = d^~ - transpose(g) * inverse(E) * d_hat
    newNx2 = data[baseIdx + 2 * nBatch] - (e * data[globalIdx] + a * data[baseIdx] + b * data[baseIdx + nBatch]);
    newNx1 = data[baseIdx + 3 * nBatch] - (d * data[globalIdx] + e * data[nBatch + globalIdx] + a * data[baseIdx + nBatch]);

    // Compute x^~ = omega * lambda
    data[baseIdx + 2 * nBatch] = omega_11 * newNx2 + omega_12 * newNx1;
    data[baseIdx + 3 * nBatch] = omega_21 * newNx2 + omega_22 * newNx1;
}