#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel2DXp (double* dataOutput, double* dataInput, const double* weights, const int numSten, const int numStenLeft, const int numStenRight, const int nxLocal, const int nyLocal, const int BLOCK_X, const int nx)
{
    extern __shared__ int memory[];

    double* arrayLocal = (double*)&memory;
    double* weightsLocal = (double*)&arrayLocal[nxLocal * nyLocal];

    // Move the weights into shared memory
    #pragma unroll
    for (int k = 0; k < numSten; k++)
    {
        weightsLocal[k] = weights[k];
    }

    // True matrix index
    int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
    int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

    // Local matrix index
    int localIdx = threadIdx.x + numStenLeft;
    int localIdy = threadIdx.y;

    double sum = 0.0;

    // Load data to shared memory with proper boundary checks
    if (globalIdx < nx && globalIdy < nx / BLOCK_X) {
        // Interior
        arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

        if (blockIdx.x != 0 && threadIdx.x < numStenLeft) {
            arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + (globalIdx - numStenLeft)];
        }

        if (blockIdx.x != nx / BLOCK_X - 1 && threadIdx.x < numStenRight) {
            arrayLocal[localIdy * nxLocal + (localIdx + BLOCK_X)] = dataInput[globalIdy * nx + globalIdx + BLOCK_X];
        }

        __syncthreads();

        // Compute the stencil
        int stenSet = localIdy * nxLocal + threadIdx.x;
        #pragma unroll
        for (int k = 0; k < numSten; k++)
        {
            sum += weightsLocal[k] * arrayLocal[stenSet + k];
        }

        // Copy results to global memory
        if (globalIdx < nx && globalIdy < nx / BLOCK_X) {
            dataOutput[globalIdy * nx + globalIdx] = sum;
        }
    }
}