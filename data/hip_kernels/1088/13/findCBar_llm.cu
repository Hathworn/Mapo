#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void findCBar(double* cOld, double* cCurr, double* cBar, int nx)
{
    // Calculate unique global thread index
    int index = blockIdx.y * blockDim.y * nx + blockIdx.x * blockDim.x + threadIdx.y * nx + threadIdx.x;
    
    // Check for within bounds before computation
    if (index < nx * gridDim.y * blockDim.y) {
        // Compute cBar using optimized access
        cBar[index] = 2.0 * cCurr[index] - cOld[index];
    }
}