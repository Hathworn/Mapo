#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void findNew(double* cCurr, double* cBar, double* cHalf, int nx)
{
    // Use a 1D grid and block configuration for better memory access patterns
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform boundary check to prevent out-of-bounds accesses
    if (index < nx * gridDim.y * blockDim.y) {
        // Compute the updated value
        cCurr[index] = cBar[index] + cHalf[index];
    }
}