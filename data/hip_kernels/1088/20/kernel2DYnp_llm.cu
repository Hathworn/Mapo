#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel2DYnp (double* dataNew, double* dataOld, double* boundaryTop, double* boundaryBottom, const double* weights, const int numSten, const int numStenTop, const int numStenBottom, const int nxLocal, const int nyLocal, const int BLOCK_Y, const int nx, const int nyTile, const int tileTop, const int tileBottom) {
    // Allocate the shared memory
    extern __shared__ int memory[];

    double* arrayLocal = (double*)&memory;
    double* weigthsLocal = (double*)&arrayLocal[nxLocal * nyLocal];

    // Move the weights into shared memory
    #pragma unroll
    for (int k = 0; k < numSten; k++) {
        weigthsLocal[k] = weights[k];
    }

    // True matrix index
    int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
    int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

    // Local matrix index
    int localIdx = threadIdx.x;
    int localIdy = threadIdx.y + numStenTop;

    // Local sum variable
    double sum = 0.0;
    int stenSet = threadIdx.y * nxLocal + localIdx; // Optimized: moved repeated calculation

    // Common logic for loading data into shared memory
    auto loadData = [&](double* topBoundary, double* bottomBoundary) {
        arrayLocal[localIdy * nxLocal + localIdx] = dataOld[globalIdy * nx + globalIdx];
        if (threadIdx.y < numStenTop) {
            arrayLocal[threadIdx.y * nxLocal + localIdx] = topBoundary[threadIdx.y * nx + globalIdx];
        }
        if (threadIdx.y < numStenBottom) {
            arrayLocal[(localIdy + BLOCK_Y) * nxLocal + localIdx] = bottomBoundary[threadIdx.y * nx + globalIdx];
        }
    };

    // Function to compute the sum based on the stencil
    auto computeSum = [&]() {
        #pragma unroll
        for (int k = 0; k < numSten; k++) {
            sum += weigthsLocal[k] * arrayLocal[stenSet + k * nxLocal];
        }
    };

    // Set all interior blocks
    if (blockIdx.y != 0 && blockIdx.y != nyTile / BLOCK_Y - 1) {
        loadData(dataOld - numStenTop * nx, dataOld + BLOCK_Y * nx);
        __syncthreads();
        computeSum();
        __syncthreads();
        dataNew[globalIdy * nx + globalIdx] = sum;
    }

    // Set all top boundary blocks
    if (blockIdx.y == 0) {
        if (tileTop != 1) {
            loadData(boundaryTop, dataOld + BLOCK_Y * nx);
            __syncthreads();
            computeSum();
            __syncthreads();
            dataNew[globalIdy * nx + globalIdx] = sum;
        } else {
            loadData(dataOld, dataOld + BLOCK_Y * nx);
            __syncthreads();
            computeSum();
            __syncthreads();
            if (threadIdx.y < BLOCK_Y - numStenTop) {
                dataNew[(globalIdy + numStenTop) * nx + globalIdx] = sum;
            }
        }
    }

    // Set the bottom boundary blocks
    if (blockIdx.y == nyTile / BLOCK_Y - 1) {
        if (tileBottom != 1) {
            loadData(dataOld - numStenTop * nx, boundaryBottom);
            __syncthreads();
            computeSum();
            __syncthreads();
            dataNew[globalIdy * nx + globalIdx] = sum;
        } else {
            loadData(dataOld - numStenTop * nx, dataOld);
            computeSum();
            __syncthreads();
            if (threadIdx.y < BLOCK_Y - numStenBottom) {
                dataNew[globalIdy * nx + globalIdx] = sum;
            }
        }
    }
}