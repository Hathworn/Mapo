#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setMultiLHS(double* dsMulti, double* dlMulti, double* diagMulti, double* duMulti, double* dwMulti, double a, double b, double c, double d, double e, int nx, int batchCount)
{
    // Calculate global index for better warp utilization
    int index = threadIdx.x + blockIdx.x * blockDim.x + (threadIdx.y + blockIdx.y * blockDim.y) * batchCount;

    // Boundary check using one condition for improved efficiency
    if (index < nx * batchCount) {
        
        // Coalesced memory accesses
        dsMulti[index] = a;
        dlMulti[index] = b;
        diagMulti[index] = c;
        duMulti[index] = d;
        dwMulti[index] = e;
    }
}