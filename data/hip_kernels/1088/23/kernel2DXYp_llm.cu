#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel2DXYp(double* dataOutput, double* dataInput, double* boundaryTop, double* boundaryBottom, const double* weights, const int numSten, const int numStenHoriz, const int numStenLeft, const int numStenRight, const int numStenVert, const int numStenTop, const int numStenBottom, const int nxLocal, const int nyLocal, const int BLOCK_X, const int BLOCK_Y, const int nx, const int nyTile)
{
    extern __shared__ double sharedMemory[];

    double* arrayLocal = sharedMemory;
    double* weightsLocal = &arrayLocal[nxLocal * nyLocal];

    // Copy weights to shared memory
    #pragma unroll
    for (int k = 0; k < numSten; k++)
    {
        weightsLocal[k] = weights[k];
    }

    int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;
    int globalIdy = blockDim.y * blockIdx.y + threadIdx.y;

    int localIdx = threadIdx.x + numStenLeft;
    int localIdy = threadIdx.y + numStenTop;

    double sum = 0.0;

    // Load data input to shared memory
    arrayLocal[localIdy * nxLocal + localIdx] = dataInput[globalIdy * nx + globalIdx];

    // Efficient boundary handling using shared memory
    if (threadIdx.x < numStenLeft)
    {
        int boundaryIdxLeft = (blockIdx.x == 0) ? nx - numStenLeft + threadIdx.x : globalIdx - numStenLeft;
        arrayLocal[localIdy * nxLocal + threadIdx.x] = dataInput[globalIdy * nx + boundaryIdxLeft];
    }

    if (threadIdx.x < numStenRight)
    {
        int boundaryIdxRight = (blockIdx.x == nx / BLOCK_X - 1) ? threadIdx.x : globalIdx + BLOCK_X;
        arrayLocal[localIdy * nxLocal + localIdx + BLOCK_X] = dataInput[globalIdy * nx + boundaryIdxRight];
    }

    if (threadIdx.y < numStenTop)
    {
        int boundaryIdxTop = (blockIdx.y == 0) ? threadIdx.y : globalIdy - numStenTop;
        arrayLocal[threadIdx.y * nxLocal + localIdx] = (blockIdx.y == 0) ? boundaryTop[threadIdx.y * nx + globalIdx] : dataInput[boundaryIdxTop * nx + globalIdx];
    }

    if (threadIdx.y < numStenBottom)
    {
        int boundaryIdxBottom = (blockIdx.y == nyTile / BLOCK_Y - 1) ? threadIdx.y : globalIdy + BLOCK_Y;
        arrayLocal[localIdy + BLOCK_Y * nxLocal + localIdx] = (blockIdx.y == nyTile / BLOCK_Y - 1) ? boundaryBottom[threadIdx.y * nx + globalIdx] : dataInput[boundaryIdxBottom * nx + globalIdx];
    }

    __syncthreads();

    int stenSet = (localIdy - numStenTop) * nxLocal + (localIdx - numStenLeft);
    int weight = 0;

    #pragma unroll
    for (int j = 0; j < numStenVert; j++)
    {
        int temp = j * nxLocal;
        #pragma unroll
        for (int i = 0; i < numStenHoriz; i++)
        {
            sum += weightsLocal[weight] * arrayLocal[stenSet + temp + i];
            weight++;
        }
    }

    dataOutput[globalIdy * nx + globalIdx] = sum;
}