#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_kernel(REAL* __restrict__ dst, REAL const value, std::size_t const count)
{
    // Calculate index using a combined declaration and computation
    std::size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for better handling of larger arrays
    for (; index < count; index += blockDim.x * gridDim.x)
    {
        dst[index] = value;
    }
}