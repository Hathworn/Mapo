#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to set an array to a specific value
__global__ void set_kernel(int* dst, int const value, int const count)
{
    // Calculate global thread index
    int const index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop to handle more elements in fewer threads
    for (int i = index; i < count; i += blockDim.x * gridDim.x)
    {
        dst[i] = value;
    }
}