#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_from_gpu()
{
    // Use shared memory to reduce redundancy
    __shared__ int bid;
    __shared__ int tid;

    // Initialize shared memory once per block
    if (threadIdx.x == 0) {
        bid = blockIdx.x;
    }
    tid = threadIdx.x;

    __syncthreads(); // Ensure block sync before using shared variables

    printf("Hello World from block %d and thread %d!\n", bid, tid);
}