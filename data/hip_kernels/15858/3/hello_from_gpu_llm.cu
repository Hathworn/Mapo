#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_from_gpu()
{
    // Use threadIdx and blockIdx to ensure each message is unique
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello World from the GPU! Thread ID: %d\n", idx);
}