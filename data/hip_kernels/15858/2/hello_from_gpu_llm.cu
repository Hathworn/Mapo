#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function with thread and block indexing
__global__ void hello_from_gpu()
{
    // Consolidated indexing
    printf("Hello World from block-%d and thread-(%d, %d)!\n", blockIdx.x, threadIdx.x, threadIdx.y);
}