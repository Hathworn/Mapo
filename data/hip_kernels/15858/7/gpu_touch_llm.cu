#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_touch(uint64_t *x, const size_t size)
{
    // Calculate global thread index
    const size_t i = blockIdx.x * blockDim.x + threadIdx.x; 
    
    // Only perform assignments if within bounds
    if (i < size)
    {
        x[i] = 0; // Set element to zero
    }
}