#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function to make use of multiple threads
__global__ void hello_from_gpu()
{
    // Use thread ID for conditional printing to demonstrate parallel execution
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid == 0) // Only let the first thread print
    {
        printf("Hello World from the GPU!\n");
    }
}