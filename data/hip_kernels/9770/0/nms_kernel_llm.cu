#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DIVUP(m,n) ((m)/(n)+((m)%(n)>0))
int const threadsPerBlock = (sizeof(unsigned long long) * 8);

__device__ inline float devIoU(float const * const a, float const * const b) {
    float left = max(a[0], b[0]), right = min(a[2], b[2]);
    float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
    float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
    float interS = width * height;
    float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
    float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
    return interS / (Sa + Sb - interS);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thres, const float *dev_boxes, unsigned long long *dev_mask) {
    const int row_start = blockIdx.y, col_start = blockIdx.x;
    
    // Stop early if block is above diagonal
    if (row_start > col_start) return;

    const int row_size = min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
    const int col_size = min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);
    
    __shared__ float block_boxes[threadsPerBlock * 5];
    
    // Load to shared memory if thread is within col_size
    if (threadIdx.x < col_size) {
        for (int j = 0; j < 5; ++j) {
            block_boxes[threadIdx.x * 5 + j] = dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 5 + j];
        }
    }
    __syncthreads();

    // Compute mask
    if (threadIdx.x < row_size) {
        const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
        const float *cur_box = dev_boxes + cur_box_idx * 5;
        unsigned long long t = 0;
        
        // Set start index for comparison
        int start = row_start == col_start ? threadIdx.x + 1 : 0;
        for (int i = start; i < col_size; i++) {
            if (devIoU(cur_box, block_boxes + i * 5) > nms_overlap_thres) {
                t |= 1ULL << i;
            }
        }

        const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
        dev_mask[cur_box_idx * col_blocks + col_start] = t;
    }
}