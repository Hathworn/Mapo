#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void non_max_supp_kernel(unsigned char *data, unsigned char *out, unsigned char *theta, int rows, int cols) {
    extern __shared__ int l_mem[];
    int* l_data = l_mem;

    const int L_SIZE = blockDim.x;
    const int g_row = blockIdx.y * blockDim.y + threadIdx.y + 1;
    const int g_col = blockIdx.x * blockDim.x + threadIdx.x + 1;
    const int l_row = threadIdx.y + 1;
    const int l_col = threadIdx.x + 1;
    const int pos = g_row * cols + g_col;

    // Load data to shared memory
    l_data[l_row * (L_SIZE + 2) + l_col] = data[pos];
    if (l_row == 1) {
        l_data[0 * (L_SIZE + 2) + l_col] = data[pos - cols];
        if (l_col == 1) l_data[0 * (L_SIZE + 2) + 0] = data[pos - cols - 1];
        else if (l_col == L_SIZE) l_data[0 * (L_SIZE + 2) + (L_SIZE + 1)] = data[pos - cols + 1];
    } else if (l_row == L_SIZE) {
        l_data[(L_SIZE + 1) * (L_SIZE + 2) + l_col] = data[pos + cols];
        if (l_col == 1) l_data[(L_SIZE + 1) * (L_SIZE + 2) + 0] = data[pos + cols - 1];
        else if (l_col == L_SIZE) l_data[(L_SIZE + 1) * (L_SIZE + 2) + (L_SIZE + 1)] = data[pos + cols + 1];
    }
    if (l_col == 1) l_data[l_row * (L_SIZE + 2) + 0] = data[pos - 1];
    else if (l_col == L_SIZE) l_data[l_row * (L_SIZE + 2) + (L_SIZE + 1)] = data[pos + 1];

    __syncthreads();

    unsigned char my_magnitude = l_data[l_row * (L_SIZE + 2) + l_col];
    
    // Improved switch-case structure
    switch (theta[pos]) {
        case 0: // Check neighbors to the East and West
            out[pos] = (my_magnitude <= l_data[l_row * (L_SIZE + 2) + l_col + 1] || 
                        my_magnitude <= l_data[l_row * (L_SIZE + 2) + l_col - 1]) ? 0 : my_magnitude;
            break;
        case 45: // Check neighbors to the NE and SW
            out[pos] = (my_magnitude <= l_data[(l_row - 1) * (L_SIZE + 2) + l_col + 1] || 
                        my_magnitude <= l_data[(l_row + 1) * (L_SIZE + 2) + l_col - 1]) ? 0 : my_magnitude;
            break;
        case 90: // Check neighbors to the North and South
            out[pos] = (my_magnitude <= l_data[(l_row - 1) * (L_SIZE + 2) + l_col] || 
                        my_magnitude <= l_data[(l_row + 1) * (L_SIZE + 2) + l_col]) ? 0 : my_magnitude;
            break;
        case 135: // Check neighbors to the NW and SE
            out[pos] = (my_magnitude <= l_data[(l_row - 1) * (L_SIZE + 2) + l_col - 1] || 
                        my_magnitude <= l_data[(l_row + 1) * (L_SIZE + 2) + l_col + 1]) ? 0 : my_magnitude;
            break;
        default: 
            out[pos] = my_magnitude; 
            break;
    }
}