#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hyst_kernel(unsigned char *data, unsigned char *out, int rows, int cols) {
    // Establish our high and low thresholds as floats
    const float lowThresh  = 10.0f;
    const float highThresh = 70.0f;

    // These variables are offset by one to avoid seg. fault errors
    // As such, this kernel ignores the outside ring of pixels
    const int row = blockIdx.y * blockDim.y + threadIdx.y + 1;
    const int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
    const int pos = row * cols + col;

    // Boundary check to avoid processing the outside ring
    if(row < rows-1 && col < cols-1) {
        const unsigned char EDGE = 255;
        const unsigned char magnitude = data[pos];

        // Process each pixel according to the threshold rules
        if(magnitude >= highThresh) {
            out[pos] = EDGE;
        } else if(magnitude <= lowThresh) {
            out[pos] = 0;
        } else {
            const float med = (highThresh + lowThresh) / 2;
            out[pos] = (magnitude >= med) ? EDGE : 0;
        }
    }
}