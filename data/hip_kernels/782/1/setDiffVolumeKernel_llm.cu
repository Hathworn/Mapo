#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setDiffVolumeKernel(float *d_fv, unsigned char *d_picture1, unsigned char *d_picture2, unsigned picWidth, unsigned picHeight) {
    __shared__ float p1_section[10 * 10 * 4];
    __shared__ float p2_section[10 * 10 * 4];
    
    // Calculate thread and block indices efficiently
    unsigned sx = threadIdx.x % 10;
    unsigned sy = threadIdx.x / 100;
    unsigned sz = (threadIdx.x % 100) / 10;

    unsigned gz = (picWidth + 9) / 10; // Simplify grid dimension calculation
    unsigned gy = (picHeight + 9) / 10;
    unsigned gx = (picWidth + 9) / 10;
    
    unsigned bz = blockIdx.x % gz;
    unsigned by = blockIdx.x / (gx * gz);
    unsigned bx = (blockIdx.x % (gx * gz)) / gz;

    unsigned vx = sx + 10 * bx;
    unsigned vy = sy + 10 * by;
    unsigned vz = sz + 10 * bz;

    // Load subpicture to shared memory with boundary checks
    if(sz == 0 && vx < picWidth && vy < picHeight) {
        for(unsigned i = 0; i < 4; i++) {
            p1_section[(sx + sy * 10) * 4 + i] = (float) d_picture1[(vx + vy * picWidth) * 4 + i];
        }
    }
    if(sx == 0 && vz < picWidth && vy < picHeight) {
        for(unsigned i = 0; i < 4; i++) {
            p2_section[(sz + sy * 10) * 4 + i] = (float) d_picture2[(vz + vy * picWidth) * 4 + i];
        }
    }
    __syncthreads();

    // Compute the difference only if within bounds
    if(vx < picWidth && vy < picHeight && vz < picWidth) {
        unsigned c1 = (sx + sy * 10) * 4;
        unsigned c2 = (sz + sy * 10) * 4;
        d_fv[vz + vx * picWidth + vy * picWidth * picWidth] =
            sqrtf(
                powf(p1_section[c1 + 0] - p2_section[c2 + 0], 2.f) +
                powf(p1_section[c1 + 1] - p2_section[c2 + 1], 2.f) +
                powf(p1_section[c1 + 2] - p2_section[c2 + 2], 2.f) +
                powf(p1_section[c1 + 3] - p2_section[c2 + 3], 2.f)
            );
    }
}