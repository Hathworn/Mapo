#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 32
#define TILE_HEIGHT 32

using namespace std;

struct matrix {
    unsigned int rows;	
    unsigned int cols;	
};

__global__ void matrix_mult(float* array1, unsigned int rows1, unsigned int cols1, float* array2, unsigned int rows2, unsigned int cols2, float* array3) {
    __shared__ float S1[TILE_WIDTH][TILE_HEIGHT];
    __shared__ float S2[TILE_HEIGHT][TILE_WIDTH];

    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int c = blockIdx.x * blockDim.x + tx;
    unsigned int r = blockIdx.y * blockDim.y + ty;
    unsigned int idx = c * rows1 + r;
    
    float val = 0;

    for (int m = 0; m < 1 + ((rows2 - 1) / TILE_WIDTH); m++) {
        int var1 = m * TILE_WIDTH + tx;
        int var2 = m * TILE_WIDTH + ty;

        // Optimize shared memory loading
        S1[ty][tx] = (r < rows1 && var1 < rows2) ? array1[r + var1 * rows1] : 0.0f;
        S2[ty][tx] = (c < cols2 && var2 < rows2) ? array2[var2 + rows2 * c] : 0.0f;

        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; i++) {
            val += S1[ty][i] * S2[i][tx];
        }

        __syncthreads();
    }

    // Store result if within bounds
    if (r < rows1 && c < cols2) {
        array3[idx] = val;
    }
}