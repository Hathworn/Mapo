#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    extern __shared__ float tmp[];
    tmp[threadIdx.x] = 0;

    // Accumulate delta values into shared memory
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            tmp[threadIdx.x] += delta[index];
        }
    }
    // Apply the scaling factor after accumulation
    mean_delta[i] = tmp[threadIdx.x] * (-1.f / sqrtf(variance[i] + .00001f));
}