#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    // Load input values and find the largest element
    for (i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = (val > largest) ? val : largest;
    }
    // Compute exponentials and their sum
    for (i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp);
        sum += e;
        output[i * stride] = e;
    }
    // Normalize output values
    for (i = 0; i < n; ++i) {
        output[i * stride] /= sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;

    int s = id % spatial;
    id = id / spatial;
    int g = id % groups;
    int b = id / groups;
    int goff = group_offset[g] * spatial;
    int boff = b * stride;
    // Invoke the device function for softmax computation
    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}