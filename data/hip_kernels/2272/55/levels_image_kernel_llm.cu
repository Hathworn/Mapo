#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    float h = hsv.x, s = hsv.y, v = hsv.z;
    float r, g, b;

    if (s == 0) {
        r = g = b = v;
    } else {
        int index = floorf(h);
        float f = h - index;
        float p = v * (1 - s);
        float q = v * (1 - s * f);
        float t = v * (1 - s * (1 - f));

        // Simplified switch case using lookup table logic
        r = (index == 0 || index == 5) ? v : (index == 1 || index == 4) ? q : p;
        g = (index == 1 || index == 2) ? v : (index == 0 || index == 3) ? t : p;
        b = (index == 3 || index == 4) ? v : (index == 2 || index == 5) ? t : q;
    }

    // Clamp color values
    return make_float3(fminf(fmaxf(r, 0.0f), 1.0f), fminf(fmaxf(g, 0.0f), 1.0f), fminf(fmaxf(b, 0.0f), 1.0f));
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    float r = rgb.x, g = rgb.y, b = rgb.z;
    float max = fmaxf(fmaxf(r, g), b);
    float min = fminf(fminf(r, g), b);
    float delta = max - min;
    
    float h = (delta == 0) ? 0 : (r == max) ? (g - b) / delta : (g == max) ? 2 + (b - r) / delta : 4 + (r - g) / delta;
    h = (h < 0) ? h + 6 : h;

    float s = (max == 0) ? 0 : delta / max;
    return make_float3(h, s, max);
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int size = batch * w * h;
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    if (id >= size) return;

    float r0 = rand[8*id + 0], r1 = rand[8*id + 1], r2 = rand[8*id + 2], r3 = rand[8*id + 3];
    saturation = r0 * (saturation - 1) + 1;
    saturation = (r1 > .5f) ? 1.f / saturation : saturation;
    exposure = r2 * (exposure - 1) + 1;
    exposure = (r3 > .5f) ? 1.f / exposure : exposure;

    float rshift = rand[0], gshift = rand[1], bshift = rand[2];
    size_t offset = id * h * w * 3;
    image += offset;
    int x = id % w, y = (id / w) % h;

    float r = image[x + w * (y + h * 0)];
    float g = image[x + w * (y + h * 1)];
    float b = image[x + w * (y + h * 2)];

    float3 rgb = make_float3(r, g, b);
    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }

    image[x + w * (y + h * 0)] = rgb.x * scale + translate + (rshift - 0.5f) * shift;
    image[x + w * (y + h * 1)] = rgb.y * scale + translate + (gshift - 0.5f) * shift;
    image[x + w * (y + h * 2)] = rgb.z * scale + translate + (bshift - 0.5f) * shift;
}