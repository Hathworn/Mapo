#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation

    if (i >= groups) return;

    float temp_sum = 0.0f; // Use register for temporary sum
    for (int k = 0; k < n; ++k) {
        temp_sum += x[k * groups + i];
    }
    
    sum[i] = temp_sum; // Store final sum
}