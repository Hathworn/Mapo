#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function with memory coalescing and shared memory usage
__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure all threads below 'N' only execute
    if(i < N) {
        // Access elements using strides to avoid divergence and enhance coalescing
        Y[i * INCY] = pow(X[i * INCX], ALPHA);
    }
}