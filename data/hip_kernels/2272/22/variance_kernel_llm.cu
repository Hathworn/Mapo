#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Calculate scale factor outside the loop
    float scale = 1.f/(batch * spatial - 1);

    // Use a single variable to capture the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float var_sum = 0; // Use a register for summation
    for(int j = 0; j < batch; ++j){
        for(int k = 0; k < spatial; ++k){
            int index = j * filters * spatial + i * spatial + k;
            float diff = x[index] - mean[i]; // Compute difference only once
            var_sum += diff * diff; // Use multiplication instead of powf
        }
    }
    variance[i] = var_sum * scale;
}