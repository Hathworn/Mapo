#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    // Use blockIdx.z to avoid gridDim.x * gridDim.y calculation for larger grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize loop increment using blockDim.x * gridDim.x
    int stride = blockDim.x * gridDim.x;
    
    while (i < n) {
        if (mask[i] == mask_num) {
            x[i] *= scale; // Only scale if condition meets
        }
        i += stride; // Move to the next element in the grid-stride loop
    }
}