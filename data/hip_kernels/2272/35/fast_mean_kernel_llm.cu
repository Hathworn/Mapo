#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    // Initialize local shared memory
    local[id] = 0;

    int i, j;
    for (j = 0; j < batch; ++j) {
        for (i = id; i < spatial; i += threads) { // Use stride loop pattern for efficiency
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += (i < spatial) ? x[index] : 0;
        }
    }

    __syncthreads();

    // Reduce shared memory values to compute the mean
    if (id == 0) {
        mean[filter] = 0;
        for (i = 0; i < threads; ++i) {
            mean[filter] += local[i];
        }
        mean[filter] /= (spatial * batch);
    }
}