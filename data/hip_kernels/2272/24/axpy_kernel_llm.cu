#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = gridDim.x * blockDim.x;
    // Use grid-stride loop for better performance
    for (; i < N; i += gridStride)
    {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}