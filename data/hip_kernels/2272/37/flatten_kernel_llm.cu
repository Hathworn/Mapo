#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    if(i >= N) return;

    // Compute indices for input and output
    int in_s = i % spatial;
    int in_c = (i / spatial) % layers;
    int b = i / (spatial * layers);
    
    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;
    
    // Conditional data transfer between arrays
    if (forward) { 
        out[i2] = x[i1];
    } else { 
        out[i1] = x[i2];
    }
}