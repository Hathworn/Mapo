#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate index using improved formula for better readability
    int i = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x;

    // Ensure that each thread checks its boundary condition
    if (i < N) X[i * INCX] = ALPHA;
}