#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation
    int stride = gridDim.x * blockDim.x; // Calculate stride for grids

    // Loop through the array using stride to maximize parallelism
    for (; i < n; i += stride) {
        if (mask[i] == mask_num) x[i] = val;
    }
}