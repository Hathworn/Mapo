#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate global thread index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x; 

    // Check if within bounds
    if(i < (NX + NY) * B) {
        int b = i / (NX + NY);
        int j = i % (NX + NY);
        
        // Use ternary operator for simplicity
        OUT[i] = (j < NX) ? X[b * NX + j] : Y[b * NY + j - NX];
    }
}