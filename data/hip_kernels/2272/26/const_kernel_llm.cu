#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Iterate over elements processed by this thread block
    int stride = blockDim.x * gridDim.x;

    for (; i < N; i += stride) {
        X[i * INCX] = ALPHA;
    }
}