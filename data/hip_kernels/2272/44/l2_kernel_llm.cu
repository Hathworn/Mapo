#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation

    // Check bounds and perform computation
    if (idx < n) {
        float diff = truth[idx] - pred[idx];
        error[idx] = diff * diff; // Error calculation
        delta[idx] = diff; // Delta calculation
    }
}