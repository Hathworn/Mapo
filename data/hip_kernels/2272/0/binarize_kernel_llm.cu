#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation for 1D grid
    if (i < n) // Use "<n" condition directly instead of ">= n" return
    {
        binary[i] = (x[i] >= 0) ? 1 : -1; // Binarize element
    }
}