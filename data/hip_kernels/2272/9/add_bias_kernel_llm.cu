#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop through all elements per thread with a stride
    for (int idx = index; idx < n * size * batch; idx += stride) {
        int i = idx % size;
        int j = (idx / size) % n;
        int k = (idx / (size * n));
        
        output[(k * n + j) * size + i] += biases[j]; // Add bias
    }
}