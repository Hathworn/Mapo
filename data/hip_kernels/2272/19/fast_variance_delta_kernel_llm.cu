#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];
    int id = threadIdx.x;
    int filter = blockIdx.x;

    // Initialize shared memory
    local[id] = 0;

    // Use unrolled loop for improved performance
    for (int j = 0; j < batch; ++j) {
        int max_i = ((spatial - 1) / threads + 1) * threads;
        for (int i = id; i < max_i; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            // Boundary check to prevent out-of-bounds access
            if (i < spatial) {
                local[id] += delta[index] * (x[index] - mean[filter]);
            }
        }
    }

    __syncthreads();

    // Use a more efficient reduction method
    if (id == 0) {
        float sum = 0.0f;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        variance_delta[filter] = sum * -.5f * powf(variance[filter] + .00001f, (float)(-3.f/2.f));
    }
}