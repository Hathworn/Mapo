#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate the global index for the thread
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    if (i < (NX + NY) * B)
    {
        // Calculate batch index and local index
        int b = i / (NX + NY);
        int j = i % (NX + NY);

        // Avoid repeated memory accesses
        float *out_val = &OUT[i]; 

        // Process X array
        if (j < NX)
        {
            if (X) atomicAdd(&X[b * NX + j], *out_val); // Use atomicAdd for concurrent updates
        }
        else // Process Y array
        {
            if (Y) atomicAdd(&Y[b * NY + j - NX], *out_val); // Use atomicAdd for concurrent updates
        }
    }
}