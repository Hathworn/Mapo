#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;

    int filter = blockIdx.x;
    local[id] = 0.0f;

    // Unroll spatial loop by half to optimize memory access
    for(int j = 0; j < batch; ++j){
        for(int i = id; i < spatial; i += 2 * threads){
            int index1 = j * spatial * filters + filter * spatial + i;
            int index2 = index1 + threads;

            local[id] += delta[index1];
            
            if(i + threads < spatial) {
                local[id] += delta[index2];
            }
        }
    }

    __syncthreads();

    // Parallel reduction to accumulate partial sums in shared memory
    for (int s = threads / 2; s > 0; s >>= 1) {
        if (id < s) {
            local[id] += local[id + s];
        }
        __syncthreads();
    }

    if (id == 0) {
        mean_delta[filter] = local[0] * (-1.f / sqrtf(variance[filter] + .00001f));
    }
}