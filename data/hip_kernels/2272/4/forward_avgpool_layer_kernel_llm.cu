#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n * c) return;

    int k = id % c;
    int b = id / c;

    float sum = 0.0f;
    for (int i = 0; i < w * h; ++i) {
        int in_index = i + h * w * (k + b * c);
        sum += input[in_index];
    }

    output[id] = sum / (w * h); // Efficiently compute average
}