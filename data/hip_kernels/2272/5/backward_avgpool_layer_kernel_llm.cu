#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified and removed gridDim.y
    if(id >= n * c) return; // Check for out of bounds

    int k = id % c;
    int b = id / c; // Use integer division to calculate b

    int out_index = k + c * b;
    float out_delta_value = out_delta[out_index] / (w * h); // Precompute shared division

    for(int i = 0; i < w * h; ++i) {
        int in_index = i + h * w * out_index; // Reuse computed out_index
        atomicAdd(&in_delta[in_index], out_delta_value); // Use atomicAdd for concurrent updates
    }
}