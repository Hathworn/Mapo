#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;

    // Use shared memory to avoid redundant global memory accesses
    extern __shared__ float temp_storage[];
    
    // Load data into shared memory
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        temp_storage[i * stride] = input[i * stride];
        atomicMax((int*)&largest, __float_as_int(temp_storage[i * stride]));
    }
    __syncthreads();

    // Calculate exponentials and sum in shared memory
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        float e = expf((temp_storage[i * stride] - largest) / temp);
        atomicAdd(&sum, e);
        temp_storage[i * stride] = e;
    }
    __syncthreads();

    // Normalize the output
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        output[i * stride] = temp_storage[i * stride] / sum;
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= batch * groups) return;
    
    int b = id / groups;
    int g = id % groups;   

    // Launch the optimized device function
    softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}