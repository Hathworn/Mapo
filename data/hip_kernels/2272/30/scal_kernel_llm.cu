#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize with cooperative grid-stride loops
__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = index; i < N; i += stride) {
        X[i * INCX] *= ALPHA;
    }
}