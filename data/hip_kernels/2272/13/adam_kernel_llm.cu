#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop over the elements with a stride to utilize all threads effectively
    for (int i = index; i < N; i += stride) {
        // Precompute factors
        float B1_pow_t = __powf(B1, t);
        float B2_pow_t = __powf(B2, t);

        float mhat = m[i] / (1.f - B1_pow_t);
        float vhat = v[i] / (1.f - B2_pow_t);

        // Update x using precomputed mhat and vhat
        x[i] = x[i] + rate * mhat / (sqrtf(vhat) + eps);
    }
}