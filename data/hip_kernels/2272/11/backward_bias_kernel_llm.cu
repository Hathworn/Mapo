#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Loop unrolling and memory coalescing
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index];
        }
    }

    part[p] = sum;
    __syncthreads();

    // Optimized reduction using parallel sum
    if (p < BLOCK / 2) {
        part[p] += part[p + BLOCK / 2];
    }
    __syncthreads();
    if (p < BLOCK / 4) {
        part[p] += part[p + BLOCK / 4];
    }
    __syncthreads();
    if (p < BLOCK / 8) {
        part[p] += part[p + BLOCK / 8];
    }
    __syncthreads();
    if (p < BLOCK / 16) {
        part[p] += part[p + BLOCK / 16];
    }
    __syncthreads();
    if (p < BLOCK / 32) {
        part[p] += part[p + BLOCK / 32];
    }
    __syncthreads();
    if (p == 0) {
        bias_updates[filter] += part[0];
    }
}