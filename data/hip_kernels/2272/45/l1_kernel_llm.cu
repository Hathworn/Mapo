#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified grid indexing
    if (i < n) {
        float diff = truth[i] - pred[i];
        error[i] = fabsf(diff); // Use faster built-in function for absolute value
        delta[i] = copysignf(1.0f, diff); // Utilize copysignf for concise conditional assignment
    }
}