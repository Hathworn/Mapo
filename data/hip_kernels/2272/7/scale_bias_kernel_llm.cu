#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if (offset < size) {
        // Precompute the bias and output index for potential performance gain
        float bias_val = biases[filter];
        int output_index = (batch * n + filter) * size + offset;
        output[output_index] *= bias_val;
    }
}