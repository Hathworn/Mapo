#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified grid indexing
    int totalThreads = blockDim.x * gridDim.x; // Total threads in grid

    for(int idx = i; idx < N; idx += totalThreads) // Loop to cover all elements
    {
        Y[idx * INCY] *= X[idx * INCX]; // Main computation
    }
}