#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Initialize a more effective index computation
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Process only within the bounds
    for (int j = idx; j < N; j += blockDim.x * gridDim.x) {
        X[j * INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[j * INCX]));
    }
}