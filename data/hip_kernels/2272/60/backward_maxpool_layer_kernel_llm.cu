#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    // Compute output dimensions
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int c = in_c;
    int area = (size - 1) / stride;

    // Calculate the thread id
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Decompose the id into indexes for batch, channel, height, width
    int index = id;
    int j = index % in_w;
    index /= in_w;
    int i = index % in_h;
    index /= in_h;
    int k = index % in_c;
    index /= in_c;
    int b = index;

    // Precalculate offsets for padding
    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    // Initialize delta to zero for accumulation
    float d = 0;

    // Loop over the pooling area
    for (int l = -area; l <= area; ++l) {
        for (int m = -area; m <= area; ++m) {
            // Compute output width and height indices
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;
            int out_index = out_w + w * (out_h + h * (k + c * b));

            // Calculate if the output position is valid
            bool valid = (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h);

            // Accumulate delta if the condition matches
            if (valid && indexes[out_index] == id)
                d += delta[out_index];
        }
    }
    // Update prev_delta
    prev_delta[id] += d;
}