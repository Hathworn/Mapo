#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Improved indexing calculation using blockIdx.x directly
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check index within bounds and update
    if(i < N) { 
        X[i * INCX] += ALPHA;
    }
}