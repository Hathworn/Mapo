#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplify thread index calculation
    int gridStride = gridDim.x * blockDim.x; // Calculate grid stride for loop

    for (int i = idx; i < n; i += gridStride) { // Use grid-stride loop for balanced load
        float common_dc = dc[i];

        if (da) da[i] += common_dc * s[i];
        if (db) db[i] += common_dc * (1 - s[i]);
        ds[i] += common_dc * (a[i] - b[i]);
    }
}