#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate unique thread index within grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Step through array by grid stride
    int gridStride = gridDim.x * blockDim.x;
    for (; i < n; i += gridStride) {
        c[i] += a[i] * b[i];
    }
}