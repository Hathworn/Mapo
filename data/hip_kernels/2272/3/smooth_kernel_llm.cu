#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Calculate indices
    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    int w_offset = -(size / 2);
    int h_offset = -(size / 2);

    int out_index = j + w * (i + h * (k + c * b));
    float out_value = x[out_index]; // Cache x[out_index] value

    int l_start = max(-h_offset, -i);
    int l_end = min(size - h_offset, h - i);
    int m_start = max(-w_offset, -j);
    int m_end = min(size - w_offset, w - j);

    for (int l = l_start; l < l_end; ++l) {
        for (int m = m_start; m < m_end; ++m) {
            int cur_h = h_offset + i + l;
            int cur_w = w_offset + j + m;
            int index = cur_w + w * (cur_h + h * (k + c * b));
            delta[out_index] += rate * (x[index] - out_value);
        }
    }
}