#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    // Calculate linear global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within bounds
    if (index < N) {
        int f = (index / spatial) % filters;
        
        // Normalize x[index]
        x[index] = (x[index] - mean[f]) / sqrtf(variance[f] + .00001f);
    }
}