#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate the global thread index more efficiently using block and thread information
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.y * gridDim.x;
  
    // Ensure that thread index is within bounds
    if (i < N)
    {
        // Perform the copying operation
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
    }
}