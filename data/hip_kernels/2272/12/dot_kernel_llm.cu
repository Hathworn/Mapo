#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Optimize block indexing
    if (index >= n * n) return; // Check for out-of-bounds
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0.0f;
    float norm1 = 0.0f;
    float norm2 = 0.0f;
    
    for(int b = 0; b < batch; ++b) {
        for(int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            sum += output[i1] * output[i2];
            norm1 += output[i1] * output[i1];
            norm2 += output[i2] * output[i2];
        }
    }
    
    float norm = sqrt(norm1) * sqrt(norm2);
    float scaled_sum = (sum / norm) * -scale;
    
    for(int b = 0; b < batch; ++b) {
        for(int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            atomicAdd(&delta[i1], scaled_sum * output[i2] / norm); // Use atomicAdd for concurrency safety
            atomicAdd(&delta[i2], scaled_sum * output[i1] / norm);
        }
    }
}