#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation

    // Unrolling loop for better memory access and performance
    int stepSize = blockDim.x * gridDim.x;  // Calculate total step size for full grid
    for (int i = idx; i < n; i += stepSize) {
        float t = truth[i];
        float p = pred[i];

        // Use conditional operator to reduce branch divergence
        error[i] = t ? -log(p) : 0.0f;  
        delta[i] = t - p;
    }
}