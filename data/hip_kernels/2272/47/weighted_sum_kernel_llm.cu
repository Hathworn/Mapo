#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Improved global index calculation by removing unnecessary components
    int totalThreads = blockDim.x * gridDim.x;      // Calculate total threads
    for (; i < n; i += totalThreads) {              // Use a loop to handle large data by iterating over the grid
        c[i] = s[i] * a[i] + (1 - s[i]) * (b ? b[i] : 0);
    }
}