#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index of the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure that the index falls within the valid range
    if(i < N) {
        float val = X[i * INCX]; // Load value once
        if(val * val < ALPHA * ALPHA) {
            X[i * INCX] = 0; // Set to zero if condition is met
        }
    }
}