#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0.0f;

    // Use a single loop with grid-stride to increment index
    for (int j = 0; j < batch * spatial; j += threads * gridDim.x) {
        int index = j + blockDim.x * blockIdx.x + id;
        if (index < batch * spatial) {
            int offset = (index / spatial) * spatial * filters + filter * spatial + (index % spatial);
            local[id] += powf(x[offset] - mean[filter], 2);
        }
    }

    __syncthreads();

    // Use parallel reduction to improve performance
    for (int stride = threads / 2; stride > 0; stride >>= 1) {
        if (id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    if (id == 0) {
        variance[filter] = local[0] / (spatial * batch - 1);
    }
}