#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Use shared memory for temporary storage to reduce global memory access
    __shared__ float s_variance_delta[1024];
    s_variance_delta[threadIdx.x] = 0;

    // Calculate variance_delta with effective memory access 
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            s_variance_delta[threadIdx.x] += delta[index] * (x[index] - mean[i]);
        }
    }

    __syncthreads(); // Synchronize to ensure all calculations are complete before writing to global memory
    
    variance_delta[i] = s_variance_delta[threadIdx.x] * -.5f * powf(variance[i] + .00001f, -1.5f);
}