#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for loop unrolling.

    for (int idx = index; idx < N; idx += stride) { // Use stride to process multiple elements per thread.
        int b = idx / spatial;
        int i = idx % spatial;
        float sum = 0;
        
        // Compute l2norm for this segment of the batch.
        for(int f = 0; f < filters; ++f) {
            int index_f = b * filters * spatial + f * spatial + i;
            sum += powf(x[index_f], 2);
        }
        sum = sqrtf(sum);
        if (sum == 0) sum = 1;

        // Normalize input and compute derivative for each filter.
        for(int f = 0; f < filters; ++f) {
            int index_f = b * filters * spatial + f * spatial + i;
            x[index_f] /= sum;
            dx[index_f] = (1 - x[index_f]) / sum;
        }
    }
}