#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCK 1024

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Loop unrolling for efficiency
    for(int b = 0; b < batch; ++b){
        for(int i = p; i < size; i += BLOCK){
            int index = i + size*(filter + n*b);
            sum += delta[index] * x_norm[index];
        }
    }

    part[p] = sum;
    __syncthreads();

    // Reduce using parallel reduction
    for (int stride = BLOCK / 2; stride > 0; stride /= 2) {
        if (p < stride) {
            part[p] += part[p + stride];
        }
        __syncthreads();
    }

    if (p == 0) {
        atomicAdd(&scale_updates[filter], part[0]);
    }
}