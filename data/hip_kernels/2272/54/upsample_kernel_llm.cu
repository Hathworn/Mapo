#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride_w = w * stride;
    size_t stride_h = h * stride;
    
    if (i >= N) return; // Early exit for threads outside of N

    int b = i / (stride_w * stride_h * c);
    int rem = i % (stride_w * stride_h * c);
    int out_c = rem / (stride_w * stride_h);
    rem = rem % (stride_w * stride_h);
    int out_h = rem / stride_w;
    int out_w = rem % stride_w;

    int in_w = out_w / stride; // Calculate input width
    int in_h = out_h / stride; // Calculate input height
    int in_index = b * w * h * c + out_c * w * h + in_h * w + in_w;

    if (forward) {
        out[i] += scale * x[in_index]; // Forward operation
    } else {
        atomicAdd(x + in_index, scale * out[i]); // Backward operation with atomic add
    }
}