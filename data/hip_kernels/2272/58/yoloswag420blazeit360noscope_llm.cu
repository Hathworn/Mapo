#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using grid-stride loop for better memory access and parallelism
__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Grid-stride loop
    for (int i = id; i < size; i += blockDim.x * gridDim.x) {
        input[i] = (rand[i] < prob) ? 0 : input[i] * scale;
    }
}