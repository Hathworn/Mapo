#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    // Calculate the global index
    int f = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Check for out-of-bounds
    if (f >= n) return;

    // Shared memory for mean calculations
    __shared__ float shared_mean[1024];

    int i = threadIdx.x;
    float local_mean = 0.0f;

    // Calculate part of the mean
    for (; i < size; i += blockDim.x) {
        local_mean += fabsf(weights[f * size + i]);
    }
    
    // Store partial sums in shared memory
    shared_mean[threadIdx.x] = local_mean;
    
    // Synchronize the threads
    __syncthreads();

    // Reduce to calculate total mean
    if (threadIdx.x == 0) {
        float total_mean = 0.0f;
        for (int j = 0; j < blockDim.x; ++j) {
            total_mean += shared_mean[j];
        }
        shared_mean[0] = total_mean / size;
    }
    
    // Synchronize again
    __syncthreads();

    // Use the computed mean to binarize weights
    float mean_value = shared_mean[0];
    for (i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? mean_value : -mean_value;
    }
}