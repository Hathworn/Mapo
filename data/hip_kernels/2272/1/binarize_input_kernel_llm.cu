#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    
    // Early exit if thread index is out of bounds
    if (s >= size) return;
    
    // Use `int` for loop index to reduce floating point operations 
    int i;
    float sum = 0;
    
    // Accumulate the absolute values
    for (i = 0; i < n; ++i) {
        sum += fabsf(input[i * size + s]);
    }
    
    // Compute mean value
    float mean = sum / n;
    
    // Assign binarized values
    for (i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}