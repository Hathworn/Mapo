#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index computation
    if (i >= filters) return;

    float sum = 0.0f; // Use local variable for accumulation, reduce global memory access
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += x[index];
        }
    }
    mean[i] = sum * scale; // Use accumulated value to calculate mean
}