#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(int *a, int dimx, int dimy) {
    // Use shared memory to reduce redundant calculations.
    __shared__ int shared_blockIdx_x, shared_threadIdx_x;  
    shared_blockIdx_x = blockIdx.x;
    shared_threadIdx_x = threadIdx.x;

    int i = shared_blockIdx_x * blockDim.x + shared_threadIdx_x;
    a[i] = shared_blockIdx_x * dimx + shared_threadIdx_x;
}