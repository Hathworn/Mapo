#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void block_sum(const float *input, float *per_block_results, const size_t n)
{
    extern __shared__ float sdata[];

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input into __shared__ memory if within bounds
    float x = (i < n) ? input[i] : 0;
    sdata[threadIdx.x] = x;
    __syncthreads();

    // Contiguous range reduction
    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if(threadIdx.x < offset)
        {
            // Add a partial sum from the "upstream" neighbor
            sdata[threadIdx.x] += sdata[threadIdx.x + offset];
        }
        // Ensure all threads in block have updated their partial sums
        __syncthreads();
    }

    // Thread 0 writes the final result
    if(threadIdx.x == 0)
    {
        per_block_results[blockIdx.x] = sdata[0];
    }
}