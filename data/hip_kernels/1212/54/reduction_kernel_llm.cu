#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction_kernel(float *g_out, float *g_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float s_data[];

    // Accumulate input with grid-stride loop into shared memory
    float input = 0.f;
    for (int i = idx_x; i < size; i += blockDim.x * gridDim.x)
    {
        input += g_in[i];
    }
    s_data[threadIdx.x] = input;

    __syncthreads();

    // Perform reduction
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x < stride)
        {
            // Use volatile to prevent unnecessary loads/stores
            volatile float *vdata = s_data;
            vdata[threadIdx.x] += vdata[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write result for the block to global memory
    if (threadIdx.x == 0) {
        g_out[blockIdx.x] = s_data[0];
    }
}