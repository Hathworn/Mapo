#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Calculate the column and row index for the C matrix
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds to avoid out-of-bounds access
    if (row < N && col < M) {
        float sum = 0.f;

        // Perform matrix multiplication
        for (int i = 0; i < K; ++i)
            sum += A[row * K + i] * B[i * M + col];

        // Write the result to C with factoring by alpha and beta
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}