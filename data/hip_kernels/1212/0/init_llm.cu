#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// HIP kernel to initialize elements of two arrays with improved indexing
__global__ void init(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory to reduce global memory access if necessary (omitted for simplicity)
    for (int i = index; i < n; i += stride) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
}