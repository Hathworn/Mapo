#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for SGEMM using shared memory
__global__ void sgemm_kernel_B(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Determine current column and row based on thread and block index
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Allocate shared memory for A and B sub-matrices
    __shared__ float shared_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float shared_B[BLOCK_SIZE][BLOCK_SIZE];

    // Initialize sum for the dot product
    float sum = 0.f;

    // Perform block-wise matrix multiplication
    for (int block = 0; block < (K + BLOCK_SIZE - 1) / BLOCK_SIZE; ++block) {
        // Load A and B data into shared memory
        if (block * BLOCK_SIZE + threadIdx.x < K && row < N)
            shared_A[threadIdx.y][threadIdx.x] = A[row * K + block * BLOCK_SIZE + threadIdx.x];
        else
            shared_A[threadIdx.y][threadIdx.x] = 0.0f;

        if (block * BLOCK_SIZE + threadIdx.y < K && col < M)
            shared_B[threadIdx.y][threadIdx.x] = B[(block * BLOCK_SIZE + threadIdx.y) * M + col];
        else
            shared_B[threadIdx.y][threadIdx.x] = 0.0f;

        // Synchronize to ensure all threads have loaded their data
        __syncthreads();

        // Perform partial dot product using loaded data
        for (int i = 0; i < BLOCK_SIZE; ++i)
            sum += shared_A[threadIdx.y][i] * shared_B[i][threadIdx.x];

        // Synchronize to ensure all threads have completed their block computations
        __syncthreads();
    }

    // Write result to C matrix with appropriate scaling
    if (row < N && col < M)
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
}
```
