#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void child_kernel(int *data, int seed)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure idx doesn't exceed bounds (additional logic can be added if bounds are known)
    if (idx < <MAX_SIZE>) { // Replace <MAX_SIZE> with actual buffer size if known
        atomicAdd(&data[idx], seed);
    }
}