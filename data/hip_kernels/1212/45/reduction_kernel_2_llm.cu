#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction_kernel_2(float *g_out, float *g_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ float s_data[];

    // Initialize shared memory with input or zero if out of bounds
    s_data[threadIdx.x] = (idx_x < size) ? g_in[idx_x] : 0.f;
    __syncthreads();

    // Optimized reduction using power of two halving
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x < stride)
        {
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        }
        __syncthreads(); // Ensure all accesses are complete before next iteration
    }

    if (threadIdx.x == 0)
    {
        g_out[blockIdx.x] = s_data[0]; // Output final reduction result
    }
}