#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan_v2_kernel(float *d_output, float *d_input, int length)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;

    extern __shared__ float s_buffer[];
    // Load input into shared memory
    int double_tid = 2 * tid;
    s_buffer[double_tid] = d_input[idx];
    s_buffer[double_tid + 1] = d_input[idx + BLOCK_DIM];

    int offset = 1;

    for (; offset < 2 * BLOCK_DIM; offset <<= 1) {
        __syncthreads();
        int idx_a = offset * (double_tid + 1) - 1;
        int idx_b = offset * (double_tid + 2) - 1;

        if (idx_b < 2 * BLOCK_DIM) {
#if (DEBUG_INDEX > 0)
            printf("[ %d, %d ]\t", idx_a, idx_b);
#endif
            s_buffer[idx_b] += s_buffer[idx_a];
        }
    }

    for (offset >>= 1; offset > 0; offset >>= 1) {
        __syncthreads();
        int idx_a = offset * (double_tid + 2) - 1;
        int idx_b = offset * (double_tid + 3) - 1;

        if (idx_b < 2 * BLOCK_DIM) {
#if (DEBUG_INDEX > 0)
            printf("[ %d, %d ]\t", idx_a, idx_b);
#endif
            s_buffer[idx_b] += s_buffer[idx_a];
        }
    }
    __syncthreads();

    // Write results to output
    d_output[idx] = s_buffer[double_tid];
    d_output[idx + BLOCK_DIM] = s_buffer[double_tid + 1];
}