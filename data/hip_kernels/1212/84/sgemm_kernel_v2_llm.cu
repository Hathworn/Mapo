#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_DIM 16

__global__ void sgemm_kernel_v2(const float *A, const float *B, float *C, int M, int N, int K, float alpha, float beta)
{
    // Get global indices
    int global_x = blockIdx.x * blockDim.x + threadIdx.x;
    int global_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Initialize local cache for the result
    float element_c = 0.f;

    // Declare shared memory for sub-matrices
    __shared__ float s_tile_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float s_tile_B[BLOCK_DIM][BLOCK_DIM];

    // Iterate over tiles
    for (int k = 0; k < K; k += BLOCK_DIM)
    {
        // Load tiles into shared memory, ensuring boundaries are not exceeded
        s_tile_A[threadIdx.y][threadIdx.x] = (global_y < M && (k + threadIdx.x) < K) ? A[global_y * K + k + threadIdx.x] : 0.0f;
        s_tile_B[threadIdx.y][threadIdx.x] = ((k + threadIdx.y) < K && global_x < N) ? B[(k + threadIdx.y) * N + global_x] : 0.0f;

        __syncthreads();

        // Compute partial product for each tile
        for (int e = 0; e < BLOCK_DIM; e++)
            element_c += s_tile_A[threadIdx.y][e] * s_tile_B[e][threadIdx.x];

        __syncthreads();
    }

    // Write the final result to global memory, ensuring within matrix boundaries
    if (global_y < M && global_x < N) {
        C[global_y * N + global_x] = alpha * element_c + beta * C[global_y * N + global_x];
    }
}