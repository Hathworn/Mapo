#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_add(int *a, int *b, int *c) {
    // Calculate the global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Perform addition
    c[idx] = a[idx] + b[idx];
}