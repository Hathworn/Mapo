#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd_kernel(float *c, const float* a, const float* b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds
    if (idx < 500) 
    {
        c[idx] = a[idx] + b[idx];
    }
}