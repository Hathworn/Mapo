#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_uniform_kernel(float *generated_out, hiprandState_t *state)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate the stride to allow threads in a block to generate multiple random numbers
    int stride = blockDim.x * gridDim.x;

    // Use loop to process more elements per thread
    for (int i = idx; i < N; i += stride)
    {
        generated_out[i] = hiprand_uniform(&state[i]);
    }
}