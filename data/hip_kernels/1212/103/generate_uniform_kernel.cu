#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generate_uniform_kernel(float *generated_out, hiprandState_t *state)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;

generated_out[idx] = hiprand_uniform(&state[idx]);
}