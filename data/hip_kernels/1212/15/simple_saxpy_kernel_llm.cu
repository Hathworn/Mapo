#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_saxpy_kernel(float *y, const float* x, const float alpha, const float beta)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index is within data bounds
    if (idx < gridDim.x * blockDim.x)
    {
        y[idx] = alpha * x[idx] + beta;
    }
}