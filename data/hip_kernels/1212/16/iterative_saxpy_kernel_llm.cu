#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iterative_saxpy_kernel(float *y, const float* x, const float alpha, const float beta, int n_loop)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure idx is within bounds of the array
    if (idx < n_loop) {
        float temp = y[idx];
        for (int i = 0; i < n_loop; i++)
            temp = alpha * x[idx] + beta;
        y[idx] = temp; // Write back the result outside the loop
    }
}