#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
    // Calculate global thread ID
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    // Perform unrolling for better memory access
    int stride = gridDim.x * blockDim.x;
    for (; i < len; i += stride) {
        out[i] = in1[i] + in2[i];
    }
}