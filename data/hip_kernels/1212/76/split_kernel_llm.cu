#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void split_kernel(float *d_output, float *d_input, float *d_predicates, float *d_scanned, int length)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Return early if the thread index exceeds the length
    if (idx >= length) return;

    // Use a predicate check to minimize branching and improve memory coalescing
    int address = d_scanned[idx] - 1;
    d_output[idx] = (d_predicates[idx] != 0.f) ? d_input[address] : 0.0f;
}