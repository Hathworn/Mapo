#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd_kernel(float *c, const float* a, const float* b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for repeated access to input arrays
    __shared__ float shared_a[blockDim.x];
    __shared__ float shared_b[blockDim.x];

    shared_a[threadIdx.x] = a[idx];
    shared_b[threadIdx.x] = b[idx];

    __syncthreads(); // Ensure input arrays are fully loaded into shared memory

    for (int i = 0; i < 500; i++)
    {
        // Read data from shared memory instead of global memory
        c[idx] = shared_a[threadIdx.x] + shared_b[threadIdx.x];
    }
}