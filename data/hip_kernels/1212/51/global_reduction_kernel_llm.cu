#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_reduction_kernel(float *data_out, float *data_in, int stride, int size)
{
    // Calculate global index and check bounds
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_stride = idx_x + stride;
    
    if (idx_stride < size) {
        // Use shared memory for faster access
        extern __shared__ float s_data[];
        s_data[threadIdx.x] = data_in[idx_x];
        __syncthreads();

        // Perform the reduction
        data_out[idx_x] += s_data[threadIdx.x + stride];

        // Synchronize threads before next stride (if necessary for correctness)
        __syncthreads();
    }
}