#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 16

__global__ void sgemm_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Shared memory for sub-matrices
    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0.f;

    // Loop over tiles
    for (int t = 0; t < (K - 1) / TILE_SIZE + 1; ++t) {
        // Load tiles into shared memory
        if (row < N && t * TILE_SIZE + threadIdx.x < K)
            sA[threadIdx.y][threadIdx.x] = A[row * K + t * TILE_SIZE + threadIdx.x];
        else
            sA[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < M && t * TILE_SIZE + threadIdx.y < K)
            sB[threadIdx.y][threadIdx.x] = B[(t * TILE_SIZE + threadIdx.y) * M + col];
        else
            sB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        // Accumulate partial results
        for (int i = 0; i < TILE_SIZE; ++i)
            sum += sA[threadIdx.y][i] * sB[i][threadIdx.x];

        __syncthreads();
    }

    // Write result to global memory
    if (row < N && col < M)
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
}