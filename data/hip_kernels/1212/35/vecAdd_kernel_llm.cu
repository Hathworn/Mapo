#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd_kernel(float *c, const float* a, const float* b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Using 'if' to check for valid index to prevent out-of-bounds access
    if (idx < 500) {
        c[idx] = a[idx] + b[idx];
    }
}