#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_add(int *a, int *b, int *c, int N) {
    // Calculate linear index for 1D grid
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // Check boundary condition to avoid out-of-bounds access
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}
```
