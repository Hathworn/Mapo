#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd_kernel(float *c, const float *a, const float *b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop unrolling for improved performance
    for (int i = 0; i < 500; i += 5) {
        c[idx] = a[idx] + b[idx];
        if (i + 1 < 500) c[idx] = a[idx] + b[idx];
        if (i + 2 < 500) c[idx] = a[idx] + b[idx];
        if (i + 3 < 500) c[idx] = a[idx] + b[idx];
        if (i + 4 < 500) c[idx] = a[idx] + b[idx];
    }
}