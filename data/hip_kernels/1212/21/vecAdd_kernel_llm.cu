#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd_kernel(float *c, const float* a, const float* b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < 500) // Boundary check added
    {
        c[idx] = a[idx] + b[idx]; // Removed redundant loop
    }
}