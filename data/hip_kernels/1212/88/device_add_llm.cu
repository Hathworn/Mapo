#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_add(int *a, int *b, int *c) {
    // Calculate the global index for the thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Perform bounds checking to prevent accessing out-of-bounds
    if (idx < gridDim.x * blockDim.x) {
        c[idx] = a[idx] + b[idx];
    }
}