#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculateHistogram(unsigned int *imageHistogram, unsigned int width, unsigned int height, hipTextureObject_t texObj)
{
    const unsigned int tidX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int tidY = blockIdx.y * blockDim.y + threadIdx.y;

    const unsigned int localId = threadIdx.y * blockDim.x + threadIdx.x;
    const unsigned int histStartIndex = (blockIdx.y * gridDim.x + blockIdx.x) * 256;

    __shared__ unsigned int histo_private[256];

    // Initialize private histogram in parallel using threads
    if (localId < 256) histo_private[localId] = 0;
    __syncthreads();

    // Read texture memory and update shared histogram atomically
    if (tidX < width && tidY < height) {
        unsigned char imageData = tex2D<unsigned char>(texObj, (float)(tidX), (float)(tidY));
        atomicAdd(&histo_private[imageData], 1);
    }

    __syncthreads();

    // Copy private histogram to global memory in parallel
    if (localId < 256) imageHistogram[histStartIndex + localId] = histo_private[localId];
}