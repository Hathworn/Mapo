#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void createResizedImage(unsigned char *imageScaledData, int scaled_width, float scale_factor, hipTextureObject_t texObj)
{
    const unsigned int tidX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int tidY = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (tidX < scaled_width && tidY < scaled_width) // Ensure index is within bounds
    {
        const unsigned index = tidY * scaled_width + tidX;

        // Read from texture memory and assign to output
        imageScaledData[index] = tex2D<unsigned char>(texObj, (float)(tidX * scale_factor), (float)(tidY * scale_factor));
    }
}