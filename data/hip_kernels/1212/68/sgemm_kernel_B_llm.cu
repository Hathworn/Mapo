#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel_B(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Use shared memory to optimize memory access
    __shared__ float Asub[32][32];
    __shared__ float Bsub[32][32];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    float sum = 0.f;

    for (int t = 0; t < (K + 31) / 32; ++t) {
        if (row < N && t * 32 + tx < K)
            Asub[ty][tx] = A[row * K + t * 32 + tx];
        else
            Asub[ty][tx] = 0.0f;
        
        if (col < M && t * 32 + ty < K)
            Bsub[ty][tx] = B[(t * 32 + ty) * M + col];
        else
            Bsub[ty][tx] = 0.0f;

        __syncthreads();

        for (int i = 0; i < 32; ++i) {
            sum += Asub[ty][i] * Bsub[i][tx];
        }
        
        __syncthreads();
    }

    if (row < N && col < M)
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
}