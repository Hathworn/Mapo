#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void idx_print()
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    int warp_idx = threadIdx.x / warpSize; // Calculate warp index
    int lane_idx = threadIdx.x & (warpSize - 1); // Calculate lane index

    if ((lane_idx & (warpSize/2 - 1)) == 0) // Condition to reduce thread divergence
        printf(" %5d\t%5d\t %2d\t%2d\n", idx, blockIdx.x, warp_idx, lane_idx); // Print thread info
}