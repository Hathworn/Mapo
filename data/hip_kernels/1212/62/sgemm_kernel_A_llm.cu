#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel_A(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    __shared__ float shared_A[32][32];  // Shared memory for tile of matrix A
    __shared__ float shared_B[32][32];  // Shared memory for tile of matrix B

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int col = blockIdx.x * blockDim.x + tx;
    int row = blockIdx.y * blockDim.y + ty;

    float sum = 0.f;

    // Loop over tiles
    for (int m = 0; m < (K + 32 - 1) / 32; ++m) {
        // Load data into shared memory
        if (m * 32 + tx < K && row < N)
            shared_A[ty][tx] = A[row * K + m * 32 + tx];
        else
            shared_A[ty][tx] = 0.f;

        if (m * 32 + ty < K && col < M)
            shared_B[ty][tx] = B[(m * 32 + ty) * M + col];
        else
            shared_B[ty][tx] = 0.f;

        __syncthreads();  // Synchronize to make sure the matrices are loaded

        // Compute partial product
        for (int e = 0; e < 32; ++e)
            sum += shared_A[ty][e] * shared_B[e][tx];

        __syncthreads();  // Synchronize before loading the new tile
    }

    if (row < N && col < M)
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
}