#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 16

__global__ void sgemm_kernel(const float *A, const float *B, float *C, int M, int N, int K, float alpha, float beta)
{
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;

    // Ensure thread is within bounds
    if (col < N && row < M)
    {
        float element_c = 0.f;
        // Use shared memory for efficient cache use
        __shared__ float Asub[TILE_SIZE][TILE_SIZE];
        __shared__ float Bsub[TILE_SIZE][TILE_SIZE];

        for (int e = 0; e < K; e += TILE_SIZE)
        {
            // Load tiles into shared memory
            Asub[threadIdx.y][threadIdx.x] = A[row * K + e + threadIdx.x];
            Bsub[threadIdx.y][threadIdx.x] = B[(e + threadIdx.y) * N + col];

            __syncthreads();

            for (int i = 0; i < TILE_SIZE; i++)
            {
                element_c += Asub[threadIdx.y][i] * Bsub[i][threadIdx.x];
            }

            __syncthreads();
        }

        // Update output matrix
        C[row * N + col] = alpha * element_c + beta * C[row * N + col];
    }
}
```
