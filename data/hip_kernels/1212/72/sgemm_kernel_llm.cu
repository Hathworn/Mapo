#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Calculate global row and column indices
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within bounds
    if (row < N && col < M) {
        float sum = 0.f;
        
        // Loop unrolling for better performance
        for (int i = 0; i < K; i += 4) {
            sum += A[row * K + i] * B[i * M + col];
            if(i + 1 < K) sum += A[row * K + (i + 1)] * B[(i + 1) * M + col];
            if(i + 2 < K) sum += A[row * K + (i + 2)] * B[(i + 2) * M + col];
            if(i + 3 < K) sum += A[row * K + (i + 3)] * B[(i + 3) * M + col];
        }

        // Update the output matrix element
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}