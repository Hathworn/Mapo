#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculateBodyForce(float4 *p, float4 *v, float dt, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;
        float4 pi = p[i]; // Cache position i to reduce global memory accesses

        for (int tile = 0; tile < gridDim.x; tile++) {
            __shared__ float3 shared_position[BLOCK_SIZE];
            float4 temp_position = p[tile * blockDim.x + threadIdx.x];
            shared_position[threadIdx.x] = make_float3(temp_position.x, temp_position.y, temp_position.z);
            __syncthreads(); // Ensure all tile data is available in shared memory

            // Use threadIdx to unroll loop partially and optimize shared memory access
            #pragma unroll 4
            for (int j = 0; j < BLOCK_SIZE; j++) {
                float dx = shared_position[j].x - pi.x;
                float dy = shared_position[j].y - pi.y;
                float dz = shared_position[j].z - pi.z;
                float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
                float invDist = rsqrtf(distSqr);
                float invDist3 = invDist * invDist * invDist;

                Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
            }
            __syncthreads(); // Synchronize before moving to the next tile
        } // End of tile loop

        v[i].x += dt * Fx; v[i].y += dt * Fy; v[i].z += dt * Fz;
    } // End of if
}