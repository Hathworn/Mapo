#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(int n, float *x, float *y) {
    int lane_id = threadIdx.x & 31;
    size_t warp_id = (threadIdx.x + blockIdx.x * blockDim.x) >> 5;
    size_t warps_per_grid = (blockDim.x * gridDim.x) >> 5;
    size_t warp_total = ((sizeof(float) * n) + STRIDE_64K - 1) / STRIDE_64K;

    // Process data by warp, with loop unrolling for efficiency
    for (; warp_id < warp_total; warp_id += warps_per_grid) {
        #pragma unroll
        for (int rep = 0; rep < STRIDE_64K / sizeof(float) / 32; rep++) {
            size_t ind = warp_id * STRIDE_64K / sizeof(float) + rep * 32 + lane_id;
            if (ind < n) {
                x[ind] = 1.0f;
                y[ind] = 2.0f;
            }
        }
    }
}