#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_kernel(unsigned int *generated_out, hiprandState_t *state)
{
    // Use shared memory for faster access
    __shared__ hiprandState_t sharedState[blockDim.x];
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize shared state
    sharedState[threadIdx.x] = state[idx];
    __syncthreads();
    
    // Generate random values
    unsigned int random_value = hiprand(&sharedState[threadIdx.x]);
    generated_out[idx] = random_value;
}