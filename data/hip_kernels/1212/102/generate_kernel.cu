#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generate_kernel(unsigned int *generated_out, hiprandState_t *state)
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;

generated_out[idx] = hiprand(&state[idx]);
}