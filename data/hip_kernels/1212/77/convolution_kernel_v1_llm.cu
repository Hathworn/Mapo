#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolution_kernel_v1(float *d_output, float *d_input, float *d_filter, int num_row, int num_col, int filter_size)
{
    // Compute thread's global position to improve coalesced memory access
    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;

    // Define shared memory for filter to reduce global memory access
    extern __shared__ float shared_filter[];

    // Load filter into shared memory
    int filter_idx = threadIdx.y * blockDim.x + threadIdx.x;
    if (filter_idx < filter_size * filter_size) {
        shared_filter[filter_idx] = d_filter[filter_idx];
    }
    __syncthreads();

    float result = 0.f;
    for (int filter_row = -filter_size / 2; filter_row <= filter_size / 2; ++filter_row)
    {
        for (int filter_col = -filter_size / 2; filter_col <= filter_size / 2; ++filter_col)
        {
            // Compute memory access indices only once per iteration
            int image_row = idx_y + filter_row;
            int image_col = idx_x + filter_col;

            // Use conditional operator to improve branch prediction
            float image_value = (image_row >= 0 && image_row < num_row && image_col >= 0 && image_col < num_col) ?
                                d_input[image_row * num_col + image_col] : 0.f;

            int filter_index = (filter_row + filter_size / 2) * filter_size + filter_col + filter_size / 2;
            result += image_value * shared_filter[filter_index];
        }
    }

    // Ensure coalesced write back to global memory
    if (idx_x < num_col && idx_y < num_row) {
        d_output[idx_y * num_col + idx_x] = result;
    }
}