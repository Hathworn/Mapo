#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction_kernel(float *g_out, float *g_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float s_data[];

    // Load data into shared memory using grid-stride loops
    float sum = 0.0f;
    for (int i = idx_x; i < size; i += blockDim.x * gridDim.x) {
        sum += g_in[i];
    }
    s_data[threadIdx.x] = sum;

    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        g_out[blockIdx.x] = s_data[0];
    }
}