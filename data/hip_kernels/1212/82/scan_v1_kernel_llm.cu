#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan_v1_kernel(float *d_output, float *d_input, int length)
{
    // Calculate the global index of the current thread
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if the index is within the bounds of the input array
    if (idx < length) {
        float element = 0.f;

        // Use shared memory for faster access
        extern __shared__ float shared_input[];

        // Load input data into shared memory
        shared_input[threadIdx.x] = d_input[idx];
        __syncthreads();

        // Perform the scan operation using shared memory
        for (int offset = 0; offset <= threadIdx.x; offset++) {
            element += shared_input[threadIdx.x - offset];
        }

        // Store the result in the output array
        d_output[idx] = element;
    }
}