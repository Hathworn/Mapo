#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd_kernel(float* c, const float* a, const float* b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Avoid repeated summation by removing the loop
    c[idx] = a[idx] + b[idx];
}