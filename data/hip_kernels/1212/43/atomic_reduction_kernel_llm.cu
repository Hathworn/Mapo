#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void atomic_reduction_kernel(float *data_out, float *data_in, int size)
{
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure that we do not access out of bounds memory
    if (idx_x < size)
    {
        atomicAdd(&data_out[0], data_in[idx_x]);
    }
}