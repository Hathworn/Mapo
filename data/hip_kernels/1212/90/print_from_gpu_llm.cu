#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void print_from_gpu(void) {
    // Combine blockIdx and threadIdx to reduce register usage
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    printf("Hello World! from thread [%d] From device\n", tid);
}