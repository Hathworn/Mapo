#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16 // Define tile width for shared memory optimization

__global__ void sgemm_kernel_B(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Define shared memory
    __shared__ float shared_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float shared_B[TILE_WIDTH][TILE_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + tx;
    int row = blockIdx.y * TILE_WIDTH + ty;

    float sum = 0.0f;

    // Loop over tiles
    for (int t = 0; t < (K + TILE_WIDTH - 1) / TILE_WIDTH; ++t) {
        // Load data into shared memory
        if (row < N && t * TILE_WIDTH + tx < K) {
            shared_A[ty][tx] = A[row * K + t * TILE_WIDTH + tx];
        } else {
            shared_A[ty][tx] = 0.0f;
        }

        if (col < M && t * TILE_WIDTH + ty < K) {
            shared_B[ty][tx] = B[(t * TILE_WIDTH + ty) * M + col];
        } else {
            shared_B[ty][tx] = 0.0f;
        }

        __syncthreads(); // Wait for all threads to finish using shared memory

        // Accumulate partial sums
        for (int i = 0; i < TILE_WIDTH; ++i) {
            sum += shared_A[ty][i] * shared_B[i][tx];
        }

        __syncthreads(); // Wait for all threads before loading new data
    }

    // Store the result back to global memory
    if (row < N && col < M) {
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}