#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_add(int *a, int *b, int *c) {
    // Pre-compute the index to reduce recalculation
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < (gridDim.x * blockDim.x)) { // Check bounds to avoid out of range access
        c[index] = a[index] + b[index];
    }
}