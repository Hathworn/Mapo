#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel_A(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Use shared memory for tiles of A and B to increase memory access efficiency
    __shared__ float tile_A[32][32];
    __shared__ float tile_B[32][32];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int col = blockIdx.x * blockDim.x + tx;
    int row = blockIdx.y * blockDim.y + ty;

    float sum = 0.f;

    // Loop over the tiles of the matrices
    for (int m = 0; m < (K + 31) / 32; ++m)
    {
        // Collaborative loading of A and B tiles into shared memory
        if (row < N && m * 32 + tx < K)
            tile_A[ty][tx] = A[row * K + m * 32 + tx];
        else
            tile_A[ty][tx] = 0.0f;

        if (col < M && m * 32 + ty < K)
            tile_B[ty][tx] = B[(m * 32 + ty) * M + col];
        else
            tile_B[ty][tx] = 0.0f;

        __syncthreads();

        // Compute the tile result
        for (int e = 0; e < 32; ++e)
            sum += tile_A[ty][e] * tile_B[e][tx];

        __syncthreads();
    }

    // Write the result back to global memory
    if (row < N && col < M)
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
}