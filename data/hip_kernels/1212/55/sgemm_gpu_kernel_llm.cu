#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_gpu_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Calculate global thread indices in x and y direction
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure threads are within matrix boundaries
    if (row < N && col < M) {
        float sum = 0.0f;

        // Use register tiling for better memory access pattern
        for (int i = 0; i < K; ++i) {
            sum += A[row * K + i] * B[i * M + col];
        }

        // Compute final element of C with alpha and beta
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}