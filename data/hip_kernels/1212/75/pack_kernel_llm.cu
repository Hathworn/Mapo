#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pack_kernel(float *d_output, float *d_input, float *d_predicates, float *d_scanned, int length)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= length) return;

    // Combine condition check and computation to reduce branch instructions
    float predicate = d_predicates[idx];
    if (predicate != 0.f)
    {
        // Precompute address before conditional statement
        int address = static_cast<int>(d_scanned[idx]) - 1;
        
        // Use conditional assignment to reduce divergent branches
        d_output[address] = d_input[idx];
    }
}