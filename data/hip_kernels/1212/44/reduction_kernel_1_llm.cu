#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction_kernel_1(float* g_out, float* g_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float s_data[];

    s_data[threadIdx.x] = (idx_x < size) ? g_in[idx_x] : 0.f;

    __syncthreads();

    // Unroll last warp; this avoids divergence and reduces syncthreads overhead
    for (unsigned int stride = blockDim.x / 2; stride > 32; stride >>= 1)
    {
        if (threadIdx.x < stride)
        {
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Process remaining warp with warp-level operations
    if (threadIdx.x < 32)
    {
        volatile float* vs_data = s_data;
        vs_data[threadIdx.x] += vs_data[threadIdx.x + 32];
        vs_data[threadIdx.x] += vs_data[threadIdx.x + 16];
        vs_data[threadIdx.x] += vs_data[threadIdx.x + 8];
        vs_data[threadIdx.x] += vs_data[threadIdx.x + 4];
        vs_data[threadIdx.x] += vs_data[threadIdx.x + 2];
        vs_data[threadIdx.x] += vs_data[threadIdx.x + 1];
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0)
        g_out[blockIdx.x] = s_data[0];
}