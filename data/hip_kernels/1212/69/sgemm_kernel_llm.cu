#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void sgemm_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Use shared memory to reduce global memory access
    __shared__ float shared_A[32][32];
    __shared__ float shared_B[32][32];

    // Calculate row and col indices
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0.f;
    
    // Load data into shared memory in tiles
    for (int tileIdx = 0; tileIdx < (K + 31) / 32; ++tileIdx)
    {
        if (tileIdx * 32 + threadIdx.x < K && row < N)
            shared_A[threadIdx.y][threadIdx.x] = A[row * K + tileIdx * 32 + threadIdx.x];
        else
            shared_A[threadIdx.y][threadIdx.x] = 0.0;

        if (tileIdx * 32 + threadIdx.y < K && col < M)
            shared_B[threadIdx.y][threadIdx.x] = B[(tileIdx * 32 + threadIdx.y) * M + col];
        else
            shared_B[threadIdx.y][threadIdx.x] = 0.0;

        // Wait for all threads to load data into shared memory
        __syncthreads();

        // Perform multiplication for this tile
        for (int k = 0; k < 32; ++k)
            sum += shared_A[threadIdx.y][k] * shared_B[k][threadIdx.x];

        // Wait before loading next tile
        __syncthreads();
    }

    // Compute final result
    if (row < N && col < M)
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
}