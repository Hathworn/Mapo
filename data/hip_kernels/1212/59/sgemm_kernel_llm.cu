#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta) 
{
    // Use shared memory for tiling
    __shared__ float Asub[32][32]; 
    __shared__ float Bsub[32][32];

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0.0f;

    for (int tileIdx = 0; tileIdx < (K + 31) / 32; ++tileIdx) {
        // Load tiles into shared memory
        if (row < N && tileIdx * 32 + threadIdx.x < K)
            Asub[threadIdx.y][threadIdx.x] = A[row * K + tileIdx * 32 + threadIdx.x];
        else
            Asub[threadIdx.y][threadIdx.x] = 0.0f;
        
        if (col < M && tileIdx * 32 + threadIdx.y < K)
            Bsub[threadIdx.y][threadIdx.x] = B[(tileIdx * 32 + threadIdx.y) * M + col];
        else
            Bsub[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads(); // Synchronize to ensure all tiles are loaded

        for (int i = 0; i < 32; ++i)
            sum += Asub[threadIdx.y][i] * Bsub[i][threadIdx.x];

        __syncthreads(); // Synchronize before loading the next tile
    }

    if (row < N && col < M)
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
}