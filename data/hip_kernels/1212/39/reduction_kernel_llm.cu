#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction_kernel(float* d_out, float* d_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float s_data[];

    // Initialize shared memory with input data or zero if out of bounds
    s_data[threadIdx.x] = (idx_x < size) ? d_in[idx_x] : 0.f;

    __syncthreads();

    // Perform reduction
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
    {
        // Check if thread is at a valid index for reduction using bitwise operation
        if ((threadIdx.x & (stride * 2 - 1)) == 0) 
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];

        // Synchronize threads before next iteration
        __syncthreads();
    }

    // Output result to global memory
    if (threadIdx.x == 0)
        d_out[blockIdx.x] = s_data[0];
}