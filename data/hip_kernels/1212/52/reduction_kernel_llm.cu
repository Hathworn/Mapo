#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction_kernel(float* d_out, float* d_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float s_data[];

    // Initialize shared memory with conditional assignment
    s_data[threadIdx.x] = (idx_x < size) ? d_in[idx_x] : 0.f;

    __syncthreads();

    // Do reduction with loop unrolling
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2)
    {
        unsigned int index = 2 * stride * threadIdx.x;
        if (threadIdx.x < blockDim.x / (2 * stride)) {
            s_data[index] += s_data[stride + index];
        }

        __syncthreads();
    }

    // Write result to global memory from the first thread
    if (threadIdx.x == 0)
        d_out[blockIdx.x] = s_data[0];
}