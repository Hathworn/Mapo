#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Calculate global thread row and column indices
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure row is within bounds
    if (row < N && col < M) {
        float sum = 0.f;
        
        // Unrolling the loop for better performance
        #pragma unroll
        for (int i = 0; i < K; ++i) {
            sum += A[row * K + i] * B[i * M + col];  // Fixed the indexing for B
        }

        // Update the matrix C with the result
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}