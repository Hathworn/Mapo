#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sgemm_kernel_A(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Calculate global thread row and column index
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds to avoid accessing out-of-range elements
    if (row < N && col < M) {
        float sum = 0.0f;

        // Use a more cache-friendly loop order
        for (int i = 0; i < K; ++i) {
            sum += A[row * K + i] * B[i * M + col]; // fix indexing error
        }

        // Write the result to the output matrix with scaling factors
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}