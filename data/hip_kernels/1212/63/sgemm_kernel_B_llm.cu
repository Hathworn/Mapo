#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel_B(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Calculate column and row indices in C
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Initialize accumulator
    float sum = 0.f;

    // Use shared memory for faster access
    __shared__ float sharedA[1024]; // Adjust size based on block size
    __shared__ float sharedB[1024]; // Adjust size based on block size

    for (int i = 0; i < K; i += blockDim.x) {
        // Load data into shared memory
        if (row < N && (i + threadIdx.x) < K) {
            sharedA[threadIdx.y * blockDim.x + threadIdx.x] = A[row * K + (i + threadIdx.x)];
        }
        if ((i + threadIdx.y) < K && col < M) {
            sharedB[threadIdx.y * blockDim.x + threadIdx.x] = B[(i + threadIdx.y) * M + col];
        }
        __syncthreads();

        for (int j = 0; j < blockDim.x; ++j) {
            sum += sharedA[threadIdx.y * blockDim.x + j] * sharedB[j * blockDim.x + threadIdx.x];
        }
        __syncthreads();
    }

    // Write result to C
    if (row < N && col < M) {
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}