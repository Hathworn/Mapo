#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_gpu_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < N && col < M) {  // Prevent out-of-bounds access
        float sum = 0.0f;
        for (int i = 0; i < K; ++i) {
            sum += A[row * K + i] * B[i * M + col];  // Correct matrix multiplication indexing
        }
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}