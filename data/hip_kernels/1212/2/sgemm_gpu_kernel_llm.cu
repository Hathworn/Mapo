#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_gpu_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within bounds
    if (col < M && row < N) {
        float sum = 0.f;
        // Loop unrolling for better performance, assuming K is a multiple of 4
        for (int i = 0; i < K; i += 4) {
            sum += A[row * K + i] * B[i * M + col];
            sum += A[row * K + i + 1] * B[(i + 1) * M + col];
            sum += A[row * K + i + 2] * B[(i + 2) * M + col];
            sum += A[row * K + i + 3] * B[(i + 3) * M + col];
        }
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}