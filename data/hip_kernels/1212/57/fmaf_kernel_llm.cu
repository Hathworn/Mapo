#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fmaf_kernel(float *d_x, float *d_y, float *d_z, int size)
{
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
  
    // Use grid-stride loop for efficient parallelism
    for (int i = idx_x; i < size; i += stride) {
        d_z[i] = __fmul_rn(d_x[i], d_y[i]); // Use fast multiplication intrinsic
    }
}