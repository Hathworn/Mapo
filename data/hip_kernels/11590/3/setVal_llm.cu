#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setVal(int* testfuck, int size) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < size) { // Ensure thread does not exceed the array bounds
        testfuck[id] = size - id;
    }
}