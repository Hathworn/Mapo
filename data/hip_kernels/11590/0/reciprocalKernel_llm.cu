#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
============================================================================
Name        : nothing.cu
Author      :
Version     :
Copyright   : Your copyright notice
Description : CUDA compute reciprocals
============================================================================
*/

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
* CUDA kernel that computes reciprocal values for a given vector
*/

/**
* Host function that copies the data and launches the work on GPU
*/
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
    // Calculate global index
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within bounds
    if (idx < vectorSize) {

        // Use precomputed reciprocal instead of division
        data[idx] = __frcp_rn(data[idx]);
    }
}