#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved empty global kernel with threadfence. 
// It ensures any previous global memory write operations 
// are visible to other threads before continuing.
__global__ void sync_conv_groups() {
    __threadfence(); // Forces memory synchronization between threads
}