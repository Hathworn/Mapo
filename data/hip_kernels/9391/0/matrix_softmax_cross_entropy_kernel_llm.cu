#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_softmax_cross_entropy_kernel(int nrow, int ncol, const float *input_a, const float *input_b, float *output) {
    extern __shared__ float loss_per_row[];
    int y = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify thread index calculation
    if (y >= nrow) return;

    input_a += y * ncol;
    input_b += y * ncol;

    // Use shared memory for maxval and sum computation
    float maxval = -FLT_MAX;
    for (int x = threadIdx.x; x < ncol; x += blockDim.x) {
        maxval = max(maxval, input_a[x]);
    }

    // Parallel reduction to find maxval per row
    __shared__ float shared_maxval;
    if (threadIdx.x == 0) shared_maxval = -FLT_MAX;
    __syncthreads();

    atomicMax(&shared_maxval, maxval);
    __syncthreads();

    float sum = 0;
    for (int x = threadIdx.x; x < ncol; x += blockDim.x) {
        sum += exp(input_a[x] - shared_maxval);
    }

    // Parallel reduction to compute sum per row
    __shared__ float shared_sum;
    if (threadIdx.x == 0) shared_sum = 0;
    __syncthreads();

    atomicAdd(&shared_sum, sum);
    __syncthreads();

    float loss = 0;
    for (int x = threadIdx.x; x < ncol; x += blockDim.x) {
        loss -= input_b[x] * log(exp(input_a[x] - shared_maxval) / shared_sum);
    }

    atomicAdd(&loss_per_row[y], loss);
    __syncthreads();

    // Use parallel reduction for mean loss across rows
    __shared__ float shared_mean_loss;
    if (threadIdx.x == 0) shared_mean_loss = 0;
    __syncthreads();

    atomicAdd(&shared_mean_loss, loss_per_row[y]);
    __syncthreads();

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        if (y == 0) {
            output[0] = shared_mean_loss / nrow;
        }
    }
}