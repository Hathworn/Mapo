#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for matrix multiplication by a constant
__global__ void matrix_multiply_by_const(const float *d_input, float *d_output, float val, int n) {
    // Calculate the global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process multiple elements per thread to optimize memory access and computation
    int stride = gridDim.x * blockDim.x;
    for (int i = index; i < n; i += stride) {
        d_output[i] = d_input[i] * val;
    }
}