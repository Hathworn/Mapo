#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_kernel(const float *input, float *output, int n) {
    // Calculate global index once and use a shared memory cache
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index < n) {
        float element = input[index];
        // Use a ternary operator for simplicity and possibly faster execution
        output[index] = element > 0 ? element : 0;
    }
}