#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel for better performance
__global__ void reduced_sum_axis_zero(const float *input_data, float *output_data, int input_n, int output_n) {
    // Calculate global index
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // Use shared memory to optimize access
    extern __shared__ float shared_sum[];
    
    if (idx < output_n) {
        float sum = 0.0f;
        // Unroll the loop to increase efficiency
        for (int i = idx; i < input_n; i += output_n) {
            sum += input_data[i];
        }
        shared_sum[threadIdx.x] = sum;
        
        // Use intra-block reduction
        __syncthreads();
        
        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                shared_sum[threadIdx.x] += shared_sum[threadIdx.x + s];
            }
            __syncthreads();
        }
        
        // Only the first thread writes back the result
        if (threadIdx.x == 0) {
            output_data[blockIdx.x] = shared_sum[0];
        }
    }
}