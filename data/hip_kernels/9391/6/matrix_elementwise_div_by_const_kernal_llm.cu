#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Optimized GPU kernel code */

// Optimized kernel function using shared memory for faster access
__global__ void matrix_elementwise_div_by_const_kernal(const float *d_in, float *d_out, float val, int n) {
    // Calculate global index
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for improved performance
    __shared__ float shared_in[256];
    
    // Copy data from global to shared memory if index is in range
    if (globalIdx < n) {
        shared_in[threadIdx.x] = d_in[globalIdx];
    }
    
    // Synchronize threads to make sure all data is copied
    __syncthreads();

    // Perform the division using shared memory
    if (globalIdx < n) {
        d_out[globalIdx] = shared_in[threadIdx.x] / val;
    }
}