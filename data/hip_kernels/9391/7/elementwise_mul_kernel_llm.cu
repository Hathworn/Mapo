#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_mul_kernel(const float *data_a, const float *data_b, float *output, int n) {
    // Calculate the global index
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;  // Calculate stride for grid-stride loop

    // Use grid-stride loop for better performance on large arrays
    for (int i = index; i < n; i += stride) {
        output[i] = data_a[i] * data_b[i];
    }
}