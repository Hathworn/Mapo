#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Optimized GPU kernel. */

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)

__global__ void matrix_elementwise_add_by_const_kernel(const float *d_in, float *d_out, float val, int n) {
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index stays within bounds
    if (index < n) {
        // Perform element-wise addition with the constant
        d_out[index] = d_in[index] + val;
    }
}