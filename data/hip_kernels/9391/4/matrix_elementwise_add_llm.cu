#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel for element-wise addition
__global__ void matrix_elementwise_add(const float *a, const float *b, float *c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Unrolled loop for better memory access pattern and performance
    int gridSize = gridDim.x * blockDim.x;
    while (index < n) {
        c[index] = a[index] + b[index];
        index += gridSize;
    }
}