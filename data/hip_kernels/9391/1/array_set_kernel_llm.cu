#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void array_set_kernel(float *array, float value, int n) {
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread ID is within bounds
    if (index < n) {
        // Assign value to the array element
        array[index] = value;
    }
}