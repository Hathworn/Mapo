#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Optimized kernel function */

__global__ void relu_gradient_kernel(const float *input, float *output, const float *in_grad, int n) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    // Use shared memory for input and output to optimize memory access
    __shared__ float shared_input[BLOCK_SIZE];
    __shared__ float shared_output[BLOCK_SIZE];
    // Load data into shared memory
    if (index < n) {
        shared_input[threadIdx.x] = input[index];
        __syncthreads();
        float element = shared_input[threadIdx.x];
        // Directly write the result to shared output memory
        shared_output[threadIdx.x] = (element > 0) ? in_grad[index] : 0;
        __syncthreads();
        output[index] = shared_output[threadIdx.x];
    }
}
```
