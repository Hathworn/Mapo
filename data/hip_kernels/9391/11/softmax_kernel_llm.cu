#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_kernel(int64_t nrow, int64_t ncol, const float *input_data, float *output_data) {
    // Calculate row index using grid-stride loop for better utilization of threads
    for (int y = blockIdx.x * blockDim.y + threadIdx.y; y < nrow; y += gridDim.x * blockDim.y) {
        const float *row_input = input_data + y * ncol;
        float *row_output = output_data + y * ncol;

        // Compute maximum value in the row
        float maxval = row_input[0];
        for (int x = 1; x < ncol; ++x) {
            maxval = max(maxval, row_input[x]);
        }

        // Compute sum of exponentials offset by max value for numerical stability
        float sum = 0;
        for (int x = 0; x < ncol; ++x) {
            sum += exp(row_input[x] - maxval);
        }

        // Compute softmax values using the calculated sum
        for (int x = 0; x < ncol; ++x) {
            row_output[x] = exp(row_input[x] - maxval) / sum;
        }
    }
}