#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void totalSequentialSharedMem(float *input, float *output, int len) {
    // Calculate the global thread index
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    // Declare shared memory with the size of blockDim.x
    __shared__ float sdata[BLOCK_SIZE];
    
    // Load input elements into shared memory and handle bounds checking
    sdata[tid] = (i < len) ? input[i] : 0.0;

    // Synchronize threads within the block to ensure all data is loaded
    __syncthreads();

    // Perform the reduction within the shared memory
    // Use a loop with logarithmic steps to reduce the data 
    // Fewer memory accesses for improved performance
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        // Synchronize threads within the block after each step of reduction
        __syncthreads();
    }

    // Output the result to global memory
    // Only the first thread in the block writes the result
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}