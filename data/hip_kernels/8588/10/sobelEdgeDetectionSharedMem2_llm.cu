#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sobelEdgeDetectionSharedMem2(int *input, int *output, int width, int height, int thresh) {

    __shared__ int sharedMem[6][6]; // Use shared memory for input window

    int i_global = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    int j_global = (blockIdx.y * blockDim.y + threadIdx.y) * 2;

    int i_shared = threadIdx.x * 2 + 1; // Local indices for shared memory
    int j_shared = threadIdx.y * 2 + 1;

    // Boundary check
    if (i_global > 0 && j_global > 0 && i_global < width - 1 && j_global < height - 1) {
        // Load input into shared memory
        for (int dy = -1; dy <= 2; ++dy) {
            for (int dx = -1; dx <= 2; ++dx) {
                sharedMem[j_shared + dy][i_shared + dx] = input[width * (j_global + dy) + i_global + dx];
            }
        }

        __syncthreads();

        int sum1 = 0, sum2 = 0, magnitude;

        for (int xind = 1; xind <= 2; ++xind) {
            for (int yind = 1; yind <= 2; ++yind) {
                sum1 = sharedMem[j_shared + yind - 1][i_shared + xind + 1] - sharedMem[j_shared + yind - 1][i_shared + xind - 1]
                     + 2 * sharedMem[j_shared + yind][i_shared + xind + 1] - 2 * sharedMem[j_shared + yind][i_shared + xind - 1]
                     + sharedMem[j_shared + yind + 1][i_shared + xind + 1] - sharedMem[j_shared + yind + 1][i_shared + xind - 1];

                sum2 = sharedMem[j_shared + yind - 1][i_shared + xind - 1] + 2 * sharedMem[j_shared + yind - 1][i_shared + xind] + sharedMem[j_shared + yind - 1][i_shared + xind + 1]
                     - sharedMem[j_shared + yind + 1][i_shared + xind - 1] - 2 * sharedMem[j_shared + yind + 1][i_shared + xind] - sharedMem[j_shared + yind + 1][i_shared + xind + 1];

                magnitude = sum1 * sum1 + sum2 * sum2;

                // Write result to output
                if (magnitude > thresh)
                    output[(j_global + yind - 1) * width + (i_global + xind - 1)] = 255;
                else
                    output[(j_global + yind - 1) * width + (i_global + xind - 1)] = 0;
            }
        }
    }
}