#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void totalWithThreadSyncAndSharedMem(float *input, float *output, int len) {
    // Use shared memory for reduction
    __shared__ float sdata[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input into shared memory and handle out of bounds
    if (i < len)
        sdata[tid] = input[i];
    else
        sdata[tid] = 0.0f;

    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int j = blockDim.x / 2; j > 0; j >>= 1) {
        if (tid < j) {
            sdata[tid] += sdata[tid + j];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}