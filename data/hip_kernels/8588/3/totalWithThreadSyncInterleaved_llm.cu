#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void totalWithThreadSyncInterleaved(float *input, float *output, int len) {
    // Get shared memory buffer
    extern __shared__ float sdata[];

    // Load input data into shared memory
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;
    if (i < len) {
        sdata[tid] = input[i];
    } else {
        sdata[tid] = 0.0f; // Prevent out-of-bounds access
    }
    __syncthreads();

    // Perform reduction within shared memory
    for (unsigned int j = 1; j < blockDim.x; j *= 2) {
        if (tid % (2 * j) == 0) {
            sdata[tid] += sdata[tid + j];
        }
        __syncthreads();
    }

    // Write result to output
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}