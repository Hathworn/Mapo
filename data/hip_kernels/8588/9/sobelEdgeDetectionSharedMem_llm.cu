#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sobelEdgeDetectionSharedMem(int *input, int *output, int width, int height, int thresh) {

    const int blockSize = 32;
    __shared__ int shMem[34][34];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= width || j >= height) return; // Boundary check

    int index = j * width + i;

    int xind = threadIdx.x + 1;
    int yind = threadIdx.y + 1;

    // Load the central tile
    shMem[xind][yind] = input[index];

    // Load the halo elements
    if (threadIdx.x == 0 && i > 0)
        shMem[xind - 1][yind] = input[width * j + i - 1];
    if (threadIdx.y == 0 && j > 0)
        shMem[xind][yind - 1] = input[width * (j - 1) + i];
    if (threadIdx.x == blockDim.x - 1 && i < width - 1)
        shMem[xind + 1][yind] = input[width * j + i + 1];
    if (threadIdx.y == blockDim.y - 1 && j < height - 1)
        shMem[xind][yind + 1] = input[width * (j + 1) + i];

    // Load the corner elements
    if (threadIdx.x == 0 && threadIdx.y == 0 && i > 0 && j > 0)
        shMem[xind - 1][yind - 1] = input[width * (j - 1) + i - 1];
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0 && i < width - 1 && j > 0)
        shMem[xind + 1][yind - 1] = input[width * (j - 1) + i + 1];
    if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1 && i > 0 && j < height - 1)
        shMem[xind - 1][yind + 1] = input[width * (j + 1) + i - 1];
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1 && i < width - 1 && j < height - 1)
        shMem[xind + 1][yind + 1] = input[width * (j + 1) + i + 1];

    __syncthreads();

    if (i > 0 && j > 0 && i < width - 1 && j < height - 1) {
        int sum1 = 0, sum2 = 0, magnitude;

        sum1 = shMem[xind + 1][yind - 1] - shMem[xind - 1][yind - 1]
            + 2 * shMem[xind + 1][yind] - 2 * shMem[xind - 1][yind]
            + shMem[xind + 1][yind + 1] - shMem[xind - 1][yind + 1];

        sum2 = shMem[xind - 1][yind - 1] + 2 * shMem[xind][yind - 1] + shMem[xind + 1][yind - 1]
            - shMem[xind - 1][yind + 1] - 2 * shMem[xind][yind + 1] - shMem[xind + 1][yind + 1];

        magnitude = sum1 * sum1 + sum2 * sum2;
        output[index] = (magnitude > thresh) ? 255 : 0;
    }
}