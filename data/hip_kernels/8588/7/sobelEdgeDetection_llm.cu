#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sobelEdgeDetection(int *input, int *output, int width, int height, int thresh) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int index = j * width + i;

    if (i > 0 && j > 0 && i < width - 1 && j < height - 1) {

        int sx = input[(j - 1) * width + (i + 1)] - input[(j - 1) * width + (i - 1)]
                 + 2 * input[j * width + (i + 1)] - 2 * input[j * width + (i - 1)]
                 + input[(j + 1) * width + (i + 1)] - input[(j + 1) * width + (i - 1)];

        int sy = input[(j - 1) * width + (i - 1)] + 2 * input[(j - 1) * width + i] + input[(j - 1) * width + (i + 1)]
                 - input[(j + 1) * width + (i - 1)] - 2 * input[(j + 1) * width + i] - input[(j + 1) * width + (i + 1)];

        int magnitude = sx * sx + sy * sy;

        // Use ternary operator for concise assignment
        output[index] = (magnitude > thresh) ? 255 : 0;
    }
}