#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sobelEdgeDetectionWithRegisters(int *input, int *output, int width, int height, int thresh) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure that the thread is within the image boundaries
    if (i > 0 && i < width - 1 && j > 0 && j < height - 1) {
        int index = j * width + i;

        // Use registers to store pixel values
        int left = input[width * j + (i - 1)];
        int right = input[width * j + (i + 1)];
        int top = input[width * (j - 1) + i];
        int bottom = input[width * (j + 1) + i];

        int val1 = input[width * (j - 1) + (i + 1)];
        int val2 = input[width * (j - 1) + (i - 1)];
        int val3 = input[width * (j + 1) + (i + 1)];
        int val4 = input[width * (j + 1) + (i - 1)];

        int sum1 = val1 - val2 + 2 * right - 2 * left + val3 - val4;
        int sum2 = val2 + 2 * top + val1 - val4 - 2 * bottom - val3;

        int magnitude = sum1 * sum1 + sum2 * sum2;
        output[index] = (magnitude > thresh) ? 255 : 0;
    } else if (i < width && j < height) {
        // For threads on the border, ensure output is set
        int index = j * width + i;
        output[index] = 0;
    }
}