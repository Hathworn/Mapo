#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void totalSequential(float *input, float *output, int len) {
    // Each thread computes part of the reduction
    extern __shared__ float sharedData[]; // Use shared memory for partial sums
    int tid = threadIdx.x;
    int globalIndex = blockIdx.x * blockDim.x + tid;

    sharedData[tid] = (globalIndex < len) ? input[globalIndex] : 0.0f;
    __syncthreads(); // Ensure all threads have written to sharedData

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sharedData[tid] += sharedData[tid + s];
        }
        __syncthreads(); // Synchronize before the next iteration
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = sharedData[0];
    }
}