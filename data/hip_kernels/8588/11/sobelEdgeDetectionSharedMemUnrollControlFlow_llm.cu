#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sobelEdgeDetectionSharedMemUnrollControlFlow(int *input, int *output, int width, int height, int thresh) {

    unsigned int blockSize = 32;
    static __shared__ int shMem[34][34];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    int xind = threadIdx.x + 1;
    int yind = threadIdx.y + 1;

    // Load the central pixel
    if (i < width && j < height) {
        shMem[xind][yind] = input[width * j + i];
    }
    
    // Load border pixels using boundary checks to avoid control flow
    if (threadIdx.x == 0 && i - 1 >= 0) {
        shMem[xind - 1][yind] = input[width * j + i - 1];
    }
    if (threadIdx.x == blockDim.x - 1 && i + 1 < width) {
        shMem[xind + 1][yind] = input[width * j + i + 1];
    }
    if (threadIdx.y == 0 && j - 1 >= 0) {
        shMem[xind][yind - 1] = input[width * (j - 1) + i];
    }
    if (threadIdx.y == blockDim.y - 1 && j + 1 < height) {
        shMem[xind][yind + 1] = input[width * (j + 1) + i];
    }
    
    __syncthreads();

    int sum1 = 0, sum2 = 0, magnitude;

    // Apply the Sobel filter only within valid pixel range
    if (i > 0 && j > 0 && i < width - 1 && j < height - 1) {
        sum1 = shMem[xind + 1][yind - 1] - shMem[xind - 1][yind - 1]
             + 2 * shMem[xind + 1][yind] - 2 * shMem[xind - 1][yind]
             + shMem[xind + 1][yind + 1] - shMem[xind - 1][yind + 1];

        sum2 = shMem[xind - 1][yind - 1] + 2 * shMem[xind][yind - 1] + shMem[xind + 1][yind - 1]
             - shMem[xind - 1][yind + 1] - 2 * shMem[xind][yind + 1] - shMem[xind + 1][yind + 1];

        magnitude = sum1 * sum1 + sum2 * sum2;

        // Threshold check with shared memory results
        output[width * j + i] = (magnitude > thresh) ? 255 : 0;
    }
}