#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Define TILESIZE constants for easy reuse
#define TILESIZE _TILESIZE_2
#define TILESIZE_SQUARE (_TILESIZE_2 * _TILESIZE_2)

// Optimized kernel for sobel edge detection
__global__ void sobelEdgeDetectionSharedMemUnrollCoalsed(int *input, int *output, int width, int height, int thresh) {
    __shared__ int shMem[4 * TILESIZE_SQUARE];

    int num = _UNROLL_;
    int size = num * TILESIZE;

    int i = blockIdx.x * (num * _TILESIZE_) + threadIdx.x;
    int j = blockIdx.y * (num * _TILESIZE_) + threadIdx.y;

    int xind = threadIdx.x;
    int yind = threadIdx.y;

    // Load data into shared memory
    for (int x = 0; x < num; x++) {
        for (int y = 0; y < num; y++) {
            int xOffset = x * (_TILESIZE_), yOffset = y * (_TILESIZE_);
            int sharedIndex = size * (yind + yOffset) + (xind + xOffset);
            int globalIndex = (j + yOffset) * width + (i + xOffset);
            shMem[sharedIndex] = input[globalIndex];
        }
    }

    __syncthreads();

    // Perform sobel operation only within valid range
    if (i < width - _TILESIZE_ && j < height - _TILESIZE_ && xind > 0 && yind > 0 && xind < (TILESIZE - 1) && yind < (TILESIZE - 1)) {
        for (int x = 0; x < num; x++) {
            for (int y = 0; y < num; y++) {
                int xOffset = x * _TILESIZE_, yOffset = y * _TILESIZE_;

                // Compute Sobel gradients
                int sum1 = shMem[(xind + 1 + xOffset) + size * (yind - 1 + yOffset)] - shMem[(xind - 1 + xOffset) + size * (yind - 1 + yOffset)]
                         + 2 * shMem[(xind + 1 + xOffset) + size * (yind + yOffset)] - 2 * shMem[(xind - 1 + xOffset) + size * (yind + yOffset)]
                         + shMem[(xind + 1 + xOffset) + size * (yind + 1 + yOffset)] - shMem[(xind - 1 + xOffset) + size * (yind + 1 + yOffset)];
                
                int sum2 = shMem[(xind - 1 + xOffset) + size * (yind - 1 + yOffset)] + 2 * shMem[(xind + xOffset) + size * (yind - 1 + yOffset)] + shMem[(xind + 1 + xOffset) + size * (yind - 1 + yOffset)]
                         - shMem[(xind - 1 + xOffset) + size * (yind + 1 + yOffset)] - 2 * shMem[(xind + xOffset) + size * (yind + 1 + yOffset)] - shMem[(xind + 1 + xOffset) + size * (yind + 1 + yOffset)];

                // Calculate magnitude
                int magnitude = sum1 * sum1 + sum2 * sum2;

                int index = (j + yOffset) * width + (i + xOffset);

                // Set output based on threshold
                output[index] = (magnitude > thresh) ? 255 : 0;
            }
        }
    }
}