#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sobelEdgeDetectionSharedMemOverlap(int *input, int *output, int width, int height, int thresh) {

    // Define shared memory
    __shared__ int shMem[_TILESIZE_2 * _TILESIZE_2];
    
    int blocksize = _TILESIZE_2;
    int i = blockIdx.x * _TILESIZE_ + threadIdx.x;
    int j = blockIdx.y * _TILESIZE_ + threadIdx.y;
    int index = j * width + i;

    int xind = threadIdx.x;
    int yind = threadIdx.y;

    // Load data to shared memory, check bounds
    if (i < width && j < height) {
        shMem[blocksize * yind + xind] = input[index];
    }
    __syncthreads();

    // Ensure valid thread indices for computation
    if(xind > 0 && yind > 0 && xind < blocksize - 1 && yind < blocksize - 1) {
        // Perform Sobel operation in shared memory
        int sum1 = shMem[xind + 1 + blocksize * (yind - 1)] - shMem[xind - 1 + blocksize * (yind - 1)]
                 + 2 * shMem[xind + 1 + blocksize * yind] - 2 * shMem[xind - 1 + blocksize * yind]
                 + shMem[xind + 1 + blocksize * (yind + 1)] - shMem[xind - 1 + blocksize * (yind + 1)];
                 
        int sum2 = shMem[xind - 1 + blocksize * (yind - 1)] + 2 * shMem[xind + blocksize * (yind - 1)] + shMem[xind + 1 + blocksize * (yind - 1)]
                 - shMem[xind - 1 + blocksize * (yind + 1)] - 2 * shMem[xind + blocksize * (yind + 1)] - shMem[xind + 1 + blocksize * (yind + 1)];
        
        int magnitude = sum1 * sum1 + sum2 * sum2;
        // Output result based on threshold
        if (i < width && j < height) {
            output[index] = (magnitude > thresh) ? 255 : 0;
        }
    }
}