#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void totalWithThreadSync(float *input, float *output, int len) {
    // Improved reduction kernel
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load shared memory/cache line to reduce global memory access
    sdata[tid] = (i < len) ? input[i] : 0.0f;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int j = blockDim.x / 2; j > 0; j >>= 1) {
        if (tid < j) {
            sdata[tid] += sdata[tid + j];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}