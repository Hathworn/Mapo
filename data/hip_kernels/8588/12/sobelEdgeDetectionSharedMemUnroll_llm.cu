#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sobelEdgeDetectionSharedMemUnroll(int *input, int *output, int width, int height, int thresh) {

    __shared__ int shMem[4 * _TILESIZE_2 * _TILESIZE_2];

    int num = _UNROLL_;
    int size = num * _TILESIZE_2;

    int i = blockIdx.x * num * _TILESIZE_ + threadIdx.x * num;
    int j = blockIdx.y * num * _TILESIZE_ + threadIdx.y * num;

    int xind = num * threadIdx.x;
    int yind = num * threadIdx.y;

    for(int x = 0; x < num; x++) {
        for(int y = 0; y < num; y++) {
            // Load input data into shared memory
            shMem[size * (yind + y) + (xind + x)] = input[(j + y) * width + (i + x)];
        }
    }

    __syncthreads();

    if (xind > 0 && yind > 0 && xind < (size - 2) && yind < (size - 2)) {
        for(int x = 0; x < num; x++) {
            for(int y = 0; y < num; y++) {
                // Precompute frequently used array indices
                int baseX = xind + x;
                int baseY = yind + y;
                int loc = baseX + size * baseY;
                
                // Calculate sum1 using precomputed indices
                int sum1 = shMem[loc + 1 + size * (-1)] - shMem[loc - 1 + size * (-1)]
                         + 2 * shMem[loc + 1] - 2 * shMem[loc - 1]
                         + shMem[loc + 1 + size] - shMem[loc - 1 + size];

                // Calculate sum2 using precomputed indices
                int sum2 = shMem[loc - 1 + size * (-1)] + 2 * shMem[loc + size * (-1)] + shMem[loc + 1 + size * (-1)]
                         - shMem[loc - 1 + size] - 2 * shMem[loc + size] - shMem[loc + 1 + size];

                int magnitude = sum1 * sum1 + sum2 * sum2;

                int index = (j + y) * width + (i + x);

                // Apply threshold
                output[index] = (magnitude > thresh) ? 255 : 0;
            }
        }
    }
}