#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void totalWithThreadSyncAndSharedMemInterleaved(float *input, float *output, int len) {
    // Utilize shared memory for block-wide reduction
    __shared__ float sdata[BLOCK_SIZE];
    
    // Calculate global index
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load input data into shared memory
    if (i < len)
        sdata[tid] = input[i];
    else
        sdata[tid] = 0.0;
    __syncthreads(); // Ensure all data is loaded before reduction begins
    
    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // Synchronize threads before next step
    }
    
    // Write result for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}