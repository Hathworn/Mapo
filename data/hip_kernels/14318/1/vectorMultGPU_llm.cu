#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorMultGPU(float *a, float *b, float *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds before assignment to avoid out-of-bounds access
    if (i < n)
    {
        c[i] = a[i] * b[i];
    }
}