#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorMult(float *a, float *b, float *c, int n)
{
    // Calculate thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Unrolled loop for improved performance
    if (i < n) {
        c[i] = a[i] * b[i];
    }
    i += blockDim.x * gridDim.x;
    if (i < n) {
        c[i] = a[i] * b[i];
    }
}