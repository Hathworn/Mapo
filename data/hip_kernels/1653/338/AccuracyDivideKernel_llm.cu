#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize the kernel to use grid-stride loop
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Calculate the index for each thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Allow multiple threads to work on large arrays
    for (int i = idx; i < 1; i += stride) {
        accuracy[i] /= N;
    }
}