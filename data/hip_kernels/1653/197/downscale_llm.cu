#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ __forceinline__ size_t idx(const size_t nc, const size_t height, const size_t width, const size_t y, const size_t x) {
    return (nc * height + y) * width + x;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    // Cache scale_factor squared to avoid recomputation in nested loops
    int sf_squared = scale_factor * scale_factor;
    
    long ii = threadIdx.x + blockDim.x * blockIdx.x;
    ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
    
    if (ii >= no_elements) return;

    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;

    for (int offset = 0; offset < sf_squared; offset++) {
        // Calculate equivalent of `i` and `j` using a single offset
        int i = offset / scale_factor;
        int j = offset % scale_factor;
        
        // Compute inverse index considering offset
        int ipidx = translate_idx_inv(threadIdx.x + blockDim.x * blockIdx.x + i * d3 + j, d1, d2, d3, scale_factor, i, j);
        
        // Accumulate results
        gradInput_data[ii] += gradOutput_data[ipidx];
    }
}