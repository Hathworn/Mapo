#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(unsigned char *ptr, int ticks) {
    // Calculate pixel position using division to reduce arithmetic operations
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y; 
    int offset = (idy * gridDim.x * blockDim.x) + idx;

    // Simplify position value calculations and reuse computed values
    int halfDim = DIM / 2;
    float fx = idx - halfDim;
    float fy = idy - halfDim;
    float distance = sqrtf(fx * fx + fy * fy);
    float cosine = cosf(distance / 10.0f - ticks / 7.0f);

    // Use constant and scalar expression for better readability and potential caching
    unsigned char grey = (unsigned char)(128.0f + (127.0f * cosine) / (distance / 10.0f + 1.0f));

    // Set pixel values efficiently
    int pixelOffset = offset * 4;
    ptr[pixelOffset] = grey;
    ptr[pixelOffset + 1] = grey;
    ptr[pixelOffset + 2] = grey;
    ptr[pixelOffset + 3] = 255;
}