#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpuDot(float* dot, float* a, float* b, int N)
{
    __shared__ float cache[THREADS_PER_BLOCK];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int cacheIdx = threadIdx.x;
    
    float temp = 0;

    // Unroll the loop to reduce loop overhead
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        temp += a[i] * b[i];
    }

    cache[cacheIdx] = temp;

    __syncthreads();

    // Optimized reduction tree
    for(int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cacheIdx < i) {
            cache[cacheIdx] += cache[cacheIdx + i];
        }
        __syncthreads();
    }

    if (cacheIdx == 0) {
        dot[blockIdx.x] = cache[0];
    }
}