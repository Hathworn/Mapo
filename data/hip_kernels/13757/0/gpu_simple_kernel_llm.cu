#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_simple_kernel(float* a, float* b, float* c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) // Correct boundary check.
        return;

    // Utilize a faster memory access pattern by reducing unnecessary I/O.
    float val_a = a[idx];
    float val_b = b[idx];
    c[idx] = val_a * val_b;

    #define PRINT_IDS
    #if !defined( __CUDA_ARCH__) || (__CUDA_ARCH__ >= 200 ) &&  defined(PRINT_IDS)
    // Print thread and block IDs for debugging purposes.
    printf("thread: %3d - block: %3d - threadIdx: %3d, warp: %3d\n", idx, blockIdx.x, threadIdx.x, threadIdx.x / warpSize);
    #endif
}