#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define threadsPerBlock 256
#define N 1024

__global__ void dot(float *a, float *b, float *c) {
    extern __shared__ float cache[]; // Use dynamic shared memory
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) { // Use for-loop for better readability
        temp += a[i] * b[i];
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i /= 2) { // Simplified loop
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}