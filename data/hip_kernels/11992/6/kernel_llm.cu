#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(unsigned char *ptr, int ticks) {
    // Calculate x, y positions
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Combine multiplications into a single offset computation
    int offset = x + y * (blockDim.x * gridDim.x);

    // Precompute reused values to avoid redundant calculations
    float fx = x - DIM / 2;
    float fy = y - DIM / 2;
    float dist = sqrtf(fx * fx + fy * fy);
    float cosineValue = cosf(dist / 10.0f - ticks / 7.0f);
    unsigned char grey = static_cast<unsigned char>(128.0f + 127.0f * cosineValue / (dist / 10.0f + 1.0f));

    // Consolidate memory writes 
    ptr[offset * 4] = grey;
    ptr[offset * 4 + 1] = grey;
    ptr[offset * 4 + 2] = grey;
    ptr[offset * 4 + 3] = 255;
}