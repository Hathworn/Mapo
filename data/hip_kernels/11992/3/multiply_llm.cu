#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiply(int a, int b, int *c) {
    // Use shared memory to reduce global memory access
    __shared__ int product;

    // Perform multiplication in shared memory
    product = a * b;

    // Store the result from shared memory to global memory
    *c = product;
}