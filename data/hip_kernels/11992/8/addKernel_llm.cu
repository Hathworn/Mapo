#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel using block and grid dimensions for larger arrays
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global index based on thread and block index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Perform addition if within bounds
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}