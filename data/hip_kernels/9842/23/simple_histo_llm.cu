#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Calculate unique index for the current thread
    int myId = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Efficiently load data from global input array
    int myItem = d_in[myId];
    
    // Determine appropriate bin for this thread's element
    int myBin = myItem % BIN_COUNT;
    
    // Use atomic operation to safely update the bin count
    atomicAdd(&d_bins[myBin], 1);
}