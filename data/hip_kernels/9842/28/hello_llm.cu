#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Pre-calculating the thread index for better readability
    int threadIndex = blockIdx.x; 

    // Utilizing printf efficiently by reducing function call overhead
    printf("Hello world! I'm a thread in block %d\n", threadIndex);
}