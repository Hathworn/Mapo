#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void remove_redness_from_coordinates(const unsigned int* d_coordinates, unsigned char* d_r, unsigned char* d_b, unsigned char* d_g, unsigned char* d_r_output, int num_coordinates, int num_pixels_y, int num_pixels_x, int template_half_height, int template_half_width) 
{
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;
    int imgSize = num_pixels_x * num_pixels_y;

    if (global_index_1d < num_coordinates) 
    {
        unsigned int image_index_1d = d_coordinates[imgSize - global_index_1d - 1];
        ushort2 image_index_2d = make_ushort2(image_index_1d % num_pixels_x, image_index_1d / num_pixels_x);

        for (int y = image_index_2d.y - template_half_height; y <= image_index_2d.y + template_half_height; y++) 
        {
            for (int x = image_index_2d.x - template_half_width; x <= image_index_2d.x + template_half_width; x++) 
            {
                // Consolidating clamping and index calculation
                int clamped_x = min(num_pixels_x - 1, max(0, x));
                int clamped_y = min(num_pixels_y - 1, max(0, y));
                int image_offset_index_1d_clamped = clamped_y * num_pixels_x + clamped_x;

                unsigned char g_value = d_g[image_offset_index_1d_clamped];
                unsigned char b_value = d_b[image_offset_index_1d_clamped];

                unsigned int gb_average = (g_value + b_value) / 2;

                d_r_output[image_offset_index_1d_clamped] = static_cast<unsigned char>(gb_average);
            }
        }
    }
}