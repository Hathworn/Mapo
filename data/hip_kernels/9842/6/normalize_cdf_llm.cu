#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
{
    // Calculate normalization constant outside of each loop to avoid redundancy
    const float normalization_constant = 1.f / d_input_cdf[n - 1];

    // Optimize global index calculation by using simple arithmetic
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    // Use direct assignment without intermediate variable for output_value
    if (global_index_1d < n)
    {
        d_output_cdf[global_index_1d] = d_input_cdf[global_index_1d] * normalization_constant;
    }
}