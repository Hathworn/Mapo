#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shmem_reduce_kernel(float *d_out, const float *d_in)
{
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // Load shared memory from global memory
    sdata[tid] = d_in[myId];
    __syncthreads();

    // Reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result of this block to global memory
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}