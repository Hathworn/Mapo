#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    for (int stage = 0; stage <= 5; stage++)
    {
        for (int substage = stage; substage >= 0; substage--)
        {
            int pair_distance = 1 << substage;
            int compare_distance = 1 << (stage + 1);

            // Calculate the index of pairs and the direction of comparison
            int pair_l = (tid / pair_distance) * pair_distance * 2 + tid % pair_distance;
            int pair_r = pair_l + pair_distance;
            int direction = ((tid & compare_distance) == 0);

            if ((pair_r < 64) && (direction == (sdata[pair_l] > sdata[pair_r])))
            {
                // Swap elements if they are in the wrong order
                float temp = sdata[pair_l];
                sdata[pair_l] = sdata[pair_r];
                sdata[pair_r] = temp;
            }
            __syncthreads(); // Synchronize between stages
        }
    }

    d_out[tid] = sdata[tid];
}