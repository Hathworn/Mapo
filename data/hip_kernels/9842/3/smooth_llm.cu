#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth(float * v_new, const float * v) {
    int myIdx = blockDim.x * blockIdx.x + threadIdx.x; // Correct thread index calculation
    int numThreads = blockDim.x * gridDim.x;
    
    // Boundary checks in case of out of bounds
    if (myIdx < numThreads) {
        int myLeftIdx = (myIdx == 0) ? 0 : myIdx - 1;
        int myRightIdx = (myIdx == (numThreads - 1)) ? numThreads - 1 : myIdx + 1;

        // Read elements
        float myElt = v[myIdx];
        float myLeftElt = v[myLeftIdx];
        float myRightElt = v[myRightIdx];

        // Smooth and write
        v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
    }
}