#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
    extern __shared__ float sdata[]; // Use shared memory
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // Load data into shared memory
    sdata[tid] = d_in[myId];
    __syncthreads();

    // Do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s]; // Perform reduction
        }
        __syncthreads(); // Ensure all adds at one stage are done
    }

    // Only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}