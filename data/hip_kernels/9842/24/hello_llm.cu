#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Using 'blockIdx.x * blockDim.x + threadIdx.x' for unique thread identification
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello world! I'm thread %d\n", idx);
}