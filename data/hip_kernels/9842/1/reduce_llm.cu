#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
    // Load input value into shared memory
    int lane = threadIdx.x % warpSize;
    s[lane] = p;

    // Perform warp-level reduction using shfl operations
    for (int offset = 1; offset < warpSize; offset *= 2) {
        unsigned int val = __shfl_down_sync(0xFFFFFFFF, s[lane], offset);
        s[lane] += val;
    }

    // Return the reduced result from the first lane
    return s[0];
}

__global__ void reduce(unsigned int * d_out_shared, const unsigned int * d_in)
{
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    if (t == 0)
    {
        *d_out_shared = sr;
    }
}