#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void maxReduce(int *d_idata, int *d_odata) {
    
    // Use a dynamically allocated shared memory
    extern __shared__ int sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x * blockDim.x + tid;

    // Load input into shared memory
    sdata[tid] = d_idata[index];
    __syncthreads();

    // Unrolling the loop for performance
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sdata[tid] = max(sdata[tid], sdata[tid + stride]);
        }
        __syncthreads(); // Ensure all memory operations are complete
    }

    // Write result for this block to output
    if (tid == 0) {
        d_odata[blockIdx.x] = sdata[0];
    }
}