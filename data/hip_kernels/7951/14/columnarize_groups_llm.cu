#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void columnarize_groups(int8_t* columnar_buffer, const int8_t* rowwise_buffer, const size_t row_count, const size_t col_count, const size_t* col_widths, const size_t row_size) {
    const auto thread_index = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure thread index is within bounds
    if (thread_index >= row_count) {
        return;
    }
    
    auto read_ptr = rowwise_buffer + thread_index * row_size;

    for (size_t i = 0; i < col_count; ++i) {
        auto col_base = columnar_buffer + i * row_count * col_widths[i];

        // Coalesce memory access for better performance
        switch (col_widths[i]) {
            case 8: {
                reinterpret_cast<int64_t*>(col_base)[thread_index] = *reinterpret_cast<const int64_t*>(read_ptr);
                break;
            }
            case 4: {
                reinterpret_cast<int32_t*>(col_base)[thread_index] = *reinterpret_cast<const int32_t*>(read_ptr);
                break;
            }
            default: {
                // Handle other column width cases if necessary
                break;
            }
        }
        
        read_ptr += col_widths[i]; // Move read pointer to the next column
    }
}