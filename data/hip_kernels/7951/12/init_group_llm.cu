#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_group(int8_t* groups, const size_t group_count, const size_t col_count, const size_t* col_widths, const size_t* init_vals) {
    const auto thread_index = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;

    if (thread_index >= group_count) {
        return;
    }

    for (size_t i = 0; i < col_count; groups += col_widths[i++] * group_count) {
        // Use shared memory for optimization
        __shared__ size_t shared_col_widths[256]; // assuming col_count won't exceed 256
        __shared__ size_t shared_init_vals[256];  // assuming col_count won't exceed 256

        if (threadIdx.x == 0) {
            shared_col_widths[i] = col_widths[i];
            shared_init_vals[i] = init_vals[i];
        }
        __syncthreads();

        switch (shared_col_widths[i]) {
            case 4:
                *reinterpret_cast<uint32_t*>(groups) = *reinterpret_cast<const uint32_t*>(shared_init_vals + i);
                break;
            case 8:
                reinterpret_cast<size_t*>(groups)[thread_index] = shared_init_vals[i];
                break;
            default:;
        }
    }
}