#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void set_valid_pos_flag(int32_t* pos_buff, const int32_t* count_buff, const int32_t entry_count) {
    const int32_t start = blockIdx.x * blockDim.x + threadIdx.x;
    const int32_t step = blockDim.x * gridDim.x;
    
    // Use shared memory for better performance if needed
    for (int32_t i = start; i < entry_count; i += step) {
        if (count_buff[i]) {
            pos_buff[i] = VALID_POS_FLAG;
        }
    }
}