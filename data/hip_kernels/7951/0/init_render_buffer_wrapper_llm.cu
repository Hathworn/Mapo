#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void init_render_buffer(int64_t* render_buffer, const uint32_t qw_count) {
    const uint32_t start = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t step = blockDim.x * gridDim.x;
    for (uint32_t i = start; i < qw_count; i += step) {
        render_buffer[i] = EMPTY_KEY_64;
    }
}

__global__ void init_render_buffer_wrapper(int64_t* render_buffer, const uint32_t qw_count) {
    // Using shared memory to avoid redundant global memory accesses
    __shared__ int64_t shared_render_buffer[256];
    const uint32_t tid = threadIdx.x;

    // Load shared memory with initial values
    if (tid < qw_count) {
        shared_render_buffer[tid] = render_buffer[tid];
    }
    __syncthreads();

    // Initialize buffer using shared memory
    init_render_buffer(shared_render_buffer, qw_count);

    __syncthreads();

    // Write back from shared memory to global memory
    if (tid < qw_count) {
        render_buffer[tid] = shared_render_buffer[tid];
    }
}