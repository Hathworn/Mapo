#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_valid_pos(int32_t* pos_buff, int32_t* count_buff, const int32_t entry_count) {
    const int32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;

    // Using local cache to reduce global memory access
    int32_t local_pos, local_count;
    for (int32_t i = idx; i < entry_count; i += stride) {
        local_pos = pos_buff[i];
        if (VALID_POS_FLAG == local_pos) {
            local_count = (i == 0) ? 0 : count_buff[i - 1];
            pos_buff[i] = local_count;
        }
    }
}