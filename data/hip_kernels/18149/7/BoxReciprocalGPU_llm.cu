#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BoxReciprocalGPU(double *gpu_prefact, double *gpu_sumRnew, double *gpu_sumInew, double *gpu_energyRecip, int imageSize)
{
    // Calculate the global thread ID
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure that threadID is within the bounds to prevent out-of-bounds access
    if(threadID < imageSize) {
        // Optimize by storing values in registers
        double sumR = gpu_sumRnew[threadID];
        double sumI = gpu_sumInew[threadID];
        double prefact = gpu_prefact[threadID];
        
        // Compute energy using values in registers for potential faster access
        gpu_energyRecip[threadID] = (sumR * sumR + sumI * sumI) * prefact;
    }
}