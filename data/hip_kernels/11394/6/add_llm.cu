#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Using block and thread indices for parallel computation
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure the index is within bounds
    if (index < 1) 
    {
        c[index] = a[index] + b[index];
    }
}