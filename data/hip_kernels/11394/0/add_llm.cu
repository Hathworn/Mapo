#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Define and implement the GPU addition function
// This version is a vector addition, with N threads
// and N blocks
// Adding one a and b instance and storing in one c instance.

// Number of blocks
#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void add(int *a, int *b, int *c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) // Ensure no out-of-bounds access
    {
        c[index] = a[index] + b[index];
    }
}