#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform calculation only up to array size
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}