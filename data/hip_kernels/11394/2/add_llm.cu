#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Use thread index for better flexibility and performance
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}