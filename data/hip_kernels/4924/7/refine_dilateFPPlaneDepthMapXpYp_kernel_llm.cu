#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refine_dilateFPPlaneDepthMapXpYp_kernel(
    float* fpPlaneDepthMap, 
    int fpPlaneDepthMap_p, 
    float* maskMap, 
    int maskMap_p, 
    int width, 
    int height, 
    int xp, 
    int yp, 
    float fpPlaneDepth) 
{
    // Calculate the absolute 2D coordinates for each thread
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check if coordinates are within the bounds of both the original and target positions
    if (x < width && y < height) 
    {
        // Calculate potential new positions
        int newX = x + xp;
        int newY = y + yp;
        
        // Verify new positions are within bounds
        if (newX >= 0 && newY >= 0 && newX < width && newY < height) 
        {
            // Read depth from maskMap and check if it's greater than zero
            float depth = maskMap[y * maskMap_p + x];
            if (depth > 0.0f) 
            {
                // Update fpPlaneDepthMap at the new position
                fpPlaneDepthMap[newY * fpPlaneDepthMap_p + newX] = fpPlaneDepth;
            }
        }
    }
}