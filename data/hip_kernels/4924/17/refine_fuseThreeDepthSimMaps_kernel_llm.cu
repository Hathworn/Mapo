#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void refine_fuseThreeDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float* isimLst, int isimLst_p, float* idptLst, int idptLst_p, float* isimAct, int isimAct_p, float* idptAct, int idptAct_p, int width, int height, float simThr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure within bounds before proceeding
    if(x >= width || y >= height)
        return;

    // Pre-calculate indexes
    int idx_odpt = y * odpt_p + x;
    int idx_idptLst = y * idptLst_p + x;
    int idx_idptAct = y * idptAct_p + x;
    int idx_osim = y * osim_p + x;
    int idx_isimLst = y * isimLst_p + x;
    int idx_isimAct = y * isimAct_p + x;

    float dpts[3] = {odpt[idx_odpt], idptLst[idx_idptLst], idptAct[idx_idptAct]};
    float sims[3] = {osim[idx_osim], isimLst[idx_isimLst], isimAct[idx_isimAct]};

    float outDpt = dpts[0];
    float outSim = simThr;

    // Use conditional assignment for efficiency
    if (sims[1] < outSim) {
        outDpt = dpts[1];
        outSim = sims[1];
    }
    if (sims[2] < outSim) {
        outDpt = dpts[2];
        outSim = sims[2];
    }

    osim[idx_osim] = outSim;
    odpt[idx_odpt] = outDpt;
}
```
