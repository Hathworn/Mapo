#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float refineDepthSubPixel(const float3& depths, const float3& sims)
{
    float outDepth = -1.0f;

    // Normalize similarities to [0, 1] range
    float simM1 = (sims.x + 1.0f) / 2.0f;
    float simP1 = (sims.z + 1.0f) / 2.0f;
    float sim1 = (sims.y + 1.0f) / 2.0f;

    if((simM1 > sim1) && (simP1 > sim1))
    {
        float dispStep = -((simP1 - simM1) / (2.0f * (simP1 + simM1 - 2.0f * sim1)));

        float floatDepthM1 = depths.x;
        float floatDepthP1 = depths.z;

        // Use pre-calculation for linear function coefficients
        float b = (floatDepthP1 + floatDepthM1) / 2.0f;
        float a = b - floatDepthM1;

        outDepth = a * dispStep + b;
    }

    return outDepth;
}

__global__ void refine_computeBestDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float3* isims, int isims_p, float3* idpts, int idpts_p, int width, int height, float simThr)
{
    // Calculate x and y based on block and thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure the coordinates are within bounds
    if((x < width) && (y < height))
    {
        float3 depths = idpts[y * idpts_p + x];
        float3 sims = isims[y * isims_p + x];

        // Select depth and similarity based on the minimum similarity
        float outDepth = ((sims.x < sims.y) ? depths.x : depths.y);
        float outSim = ((sims.x < sims.y) ? sims.x : sims.y);
        outDepth = ((sims.z < outSim) ? depths.z : outDepth);
        outSim = ((sims.z < outSim) ? sims.z : outSim);

        // Refine depth if possible
        float refinedDepth = refineDepthSubPixel(depths, sims);
        if(refinedDepth > 0.0f)
        {
            outDepth = refinedDepth;
        }

        // Update similarity and depth maps with threshold check
        osim[y * osim_p + x] = (outSim < simThr ? outSim : 1.0f);
        odpt[y * odpt_p + x] = (outSim < simThr ? outDepth : -1.0f);
    }
}