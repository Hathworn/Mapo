#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generateGaussian_kernel(float* og, float delta, int radius)
{
    // Cache the thread ID and pre-calculate repeated values
    int tid = threadIdx.x;
    float invTwoDeltaSquared = 1.0f / (2.0f * delta * delta);

    // Optimize x calculation to reduce operations
    int x = tid - radius;

    // Use fast math operation for exponent calculation
    og[tid] = __expf(-(x * x) * invTwoDeltaSquared);
}