#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiplyTiled(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int col = blockIdx.x * TILE_WIDTH + tx;
    unsigned int row = blockIdx.y * TILE_WIDTH + ty;
    float acc = 0;

    for (int t = 0; t < (numAColumns - 1) / TILE_WIDTH + 1; ++t) {
        unsigned int ATilePitch = t * TILE_WIDTH + tx;
        unsigned int BTilePitch = t * TILE_WIDTH + ty;

        // Use conditional operators to avoid branching
        ds_A[ty][tx] = (row < numARows && ATilePitch < numAColumns) ? A[row * numAColumns + ATilePitch] : 0;
        ds_B[ty][tx] = (col < numBColumns && BTilePitch < numBRows) ? B[BTilePitch * numBColumns + col] : 0;

        __syncthreads();
        #pragma unroll
        for (int k = 0; k < TILE_WIDTH; ++k) {
            acc += ds_A[ty][k] * ds_B[k][tx];
        }
        __syncthreads();
    }

    // Reduce arithmetic for boundary check
    if (row < numCRows && col < numCColumns) {
        C[row * numCColumns + col] = acc;
    }
}