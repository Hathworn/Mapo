#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < len) {
        out[i] = __fadd_rn(in1[i], in2[i]); // Use fast intrinsic function for addition
    }
}