#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan(float* in, float* out, float* post, int len) {
    __shared__ float scan_array[HALF_BLOCK_SIZE];
    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * BLOCK_SIZE;
    
    // Load elements into shared memory with boundary checks
    scan_array[t] = (start + t < len) ? in[start + t] : 0;
    scan_array[BLOCK_SIZE + t] = (start + BLOCK_SIZE + t < len) ? in[start + BLOCK_SIZE + t] : 0;
    __syncthreads();
    
    // Up-sweep/reduction phase
    for (unsigned int stride = 1; stride <= BLOCK_SIZE; stride <<= 1) {
        int index = (t + 1) * stride * 2 - 1;
        if (index < 2 * BLOCK_SIZE) scan_array[index] += scan_array[index - stride];
        __syncthreads();
    }
    
    // Down-sweep phase
    if (t == 0) scan_array[2 * BLOCK_SIZE - 1] = 0;
    __syncthreads();

    for (unsigned int stride = BLOCK_SIZE; stride > 0; stride >>= 1) {
        int index = (t + 1) * stride * 2 - 1;
        if (index + stride < 2 * BLOCK_SIZE) {
            float temp = scan_array[index];
            scan_array[index] += scan_array[index + stride];
            scan_array[index + stride] = temp;
        }
        __syncthreads();
    }

    // Write results back to global memory
    if (start + t < len) out[start + t] = scan_array[t];
    if (start + BLOCK_SIZE + t < len) out[start + BLOCK_SIZE + t] = scan_array[BLOCK_SIZE + t];
    
    // Save block's aggregate result if post array is provided
    if (post && t == 0) post[blockIdx.x] = scan_array[2 * BLOCK_SIZE - 1];
}