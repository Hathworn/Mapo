#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 1024

#ifndef THREADS
# define THREADS 1024
#endif

__global__ void total(float * input, float * output, unsigned int len) {
    __shared__ float sum[2 * BLOCK_SIZE];
    unsigned int i = threadIdx.x;
    unsigned int j = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    // Initialize the local sum with bounds check
    float localSum = 0.0f;
    if (j < len) localSum = input[j];
    if (j + blockDim.x < len) localSum += input[j + blockDim.x];

    // Store local sum into shared memory
    sum[i] = localSum;
    __syncthreads();

    // Efficient reduction in shared memory
    for (unsigned int step = blockDim.x / 2; step > 0; step >>= 1) {
        if (i < step) {
            sum[i] = localSum = localSum + sum[i + step];
        }
        __syncthreads();
    }

    // Store result of reduction
    if (i == 0) output[blockIdx.x] = sum[0];
}