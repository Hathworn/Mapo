#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void post_scan(float* in, float* add, int len) {
    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * BLOCK_SIZE;

    // Load previous block sum only once
    float blockAdd = (blockIdx.x) ? add[blockIdx.x - 1] : 0;

    // Check boundaries and apply addition
    if (start + t < len) {
        in[start + t] += blockAdd;
    }
    if (start + BLOCK_SIZE + t < len) {
        in[start + BLOCK_SIZE + t] += blockAdd;
    }
}