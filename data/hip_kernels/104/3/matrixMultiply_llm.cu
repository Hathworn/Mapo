#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Define TILE WIDTH for shared memory optimization
#define TILE_WIDTH 16

__global__ void matrixMultiply(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    // Define shared memory tiles
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float value = 0;

    // Loop over tiles j
    for (int j = 0; j < (numAColumns - 1) / TILE_WIDTH + 1; ++j) {
        // Load data into shared memory
        if (row < numARows && j * TILE_WIDTH + threadIdx.x < numAColumns)
            tileA[threadIdx.y][threadIdx.x] = A[row * numAColumns + j * TILE_WIDTH + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0;

        if (col < numBColumns && j * TILE_WIDTH + threadIdx.y < numBRows)
            tileB[threadIdx.y][threadIdx.x] = B[(j * TILE_WIDTH + threadIdx.y) * numBColumns + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0;

        // Synchronize threads
        __syncthreads();

        // Multiply and accumulate the results
        #pragma unroll
        for (int k = 0; k < TILE_WIDTH; ++k)
            value += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];

        // Synchronize threads
        __syncthreads();
    }

    // Write results to C
    if ((row < numCRows) && (col < numCColumns))
        C[row * numCColumns + col] = value;
}