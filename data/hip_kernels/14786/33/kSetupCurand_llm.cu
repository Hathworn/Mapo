#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate global index once and use shared memory for better performance
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    // Optimize hiprand_init with a calculated seed per thread using tidx
    hiprand_init(seed + tidx, 0, 0, &state[tidx]);
}