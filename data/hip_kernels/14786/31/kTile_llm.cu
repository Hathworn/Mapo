#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate the global index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate the number of threads for the whole grid
    const int numThreads = blockDim.x * gridDim.x;

    // Loop over target array elements using stride of total number of threads
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Calculate 2D target coordinates
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;

        // Map target coordinates to source coordinates using modulo operation
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Map source element to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}