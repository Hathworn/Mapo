#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numCols, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];
    
    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Unrolling loop for better performance
    if (eidx < numCols) {
        for (; eidx < numElements; eidx += numCols) {
            sum += a[eidx] * b[eidx];
        }
    }
    
    shmem[threadIdx.x] = sum;
    __syncthreads();
    
    // Optimize the reduction by using a single warp
    if (blockDim.x >= 512) { if (threadIdx.x < 256) shmem[threadIdx.x] += shmem[threadIdx.x + 256]; __syncthreads(); }
    if (blockDim.x >= 256) { if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128]; __syncthreads(); }
    if (blockDim.x >= 128) { if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64]; __syncthreads(); }
    
    // Use warp shuffle for final warp reduction
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }
    
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}