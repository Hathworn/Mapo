#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RgbToRgbaKernel(const uint8_t *__restrict__ input, uint8_t *__restrict__ output, size_t pitch, size_t width_px, size_t height) {
    constexpr size_t in_channels = 3, out_channels = 4;
    size_t x = threadIdx.x + blockIdx.x * blockDim.x;
    size_t y = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Early exit for threads outside image bounds
    if (x >= width_px || y >= height) return;
    
    size_t in_idx = in_channels * x + in_channels * width_px * y;
    size_t out_idx = out_channels * x + pitch * y;
    
    // Coalesce memory accesses for better cache efficiency
    uint8_t r = input[in_idx];
    uint8_t g = input[in_idx + 1];
    uint8_t b = input[in_idx + 2];
    
    output[out_idx] = r;
    output[out_idx + 1] = g;
    output[out_idx + 2] = b;
    output[out_idx + 3] = 255;  // Set alpha channel to maximum
}