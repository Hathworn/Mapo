#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ConvertGrayToYCbCr8uKernel(const uint8_t *input, uint8_t *output, unsigned int total_pixels) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total_pixels) {
        return;
    }

    /* Use `C` macro instead of hardcoding and precompute index */
    #define C 3
    unsigned int out_idx = idx * C;
    
    const uint8_t pixel_in = input[idx];
    output[out_idx] = pixel_in;
    output[out_idx + 1] = 128;
    output[out_idx + 2] = 128;
}