#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory for cooperative access
    __shared__ float shared_accuracy;
    
    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy;
    }
    __syncthreads();
    
    // Each thread divides the shared accuracy once
    shared_accuracy /= N;
    
    __syncthreads();
    
    if (threadIdx.x == 0) {
        *accuracy = shared_accuracy;
    }
}