#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Calculate global thread ID
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop to ensure all elements are processed
    for (int idx = pos; idx < len; idx += blockDim.x * gridDim.x) {
        d_res[idx] = d_src[idx] * scale;
    }
}