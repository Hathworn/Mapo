#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Utilize shared memory for faster access
    extern __shared__ short2 sharedLoc[];
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Load data into shared memory
        sharedLoc[threadIdx.x] = loc_[ptidx];
        __syncthreads();

        // Use shared memory values
        x[ptidx] = sharedLoc[threadIdx.x].x * scale;
        y[ptidx] = sharedLoc[threadIdx.x].y * scale;
    }
}