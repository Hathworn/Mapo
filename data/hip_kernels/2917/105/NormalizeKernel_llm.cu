#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float* __restrict__ normalization_factor, int w, int h, int s, float* __restrict__ image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < h && j < w) // Reorder condition to enhance readability
    {
        const int pos = i * s + j;
        float scale = normalization_factor[pos];
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);
        image[pos] *= invScale;
    }
}