#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Early exit if out of bounds
    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate bottom left corner of a target pixel using flow
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // Fractional and integral decomposition
    float px, py;
    float dx = modff (cx, &px);
    float dy = modff (cy, &py);

    // Target pixel coordinates
    int tx = (int) px;
    int ty = (int) py;

    float value = src[image_row_offset + j];
    float weight;

    // Utilize inline function for atomic add
    auto atomic_add_wrapper = [](float* address, float value) {
        atomicAdd(address, value);
    };

    // Avoid repeated calculations for checks
    auto addToPixel = [&](int x, int y, float weight_val) {
        if ((x >= 0) && (x < w) && (y >= 0) && (y < h)) {
            atomic_add_wrapper(dst + y * image_stride + x, value * weight_val);
            atomic_add_wrapper(normalization_factor + y * image_stride + x, weight_val);
        }
    };

    // Fill pixels based on barycentric coordinates
    addToPixel(tx, ty, dx * dy);
    tx -= 1;
    addToPixel(tx, ty, (1.0f - dx) * dy);
    ty -= 1;
    addToPixel(tx, ty, (1.0f - dx) * (1.0f - dy));
    tx += 1;
    addToPixel(tx, ty, dx * (1.0f - dy));
}