#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii /= d3;
    z = ii % d2;
    ii /= d2;
    y = ii % d1;
    ii /= d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii /= d3;
    z = ii % d2;
    ii /= d2;
    y = ii % d1;
    ii /= d1;
    x = ii;
    w /= scale_factor;
    z /= scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ __forceinline__ size_t idx(const size_t nc, const size_t height, const size_t width, const size_t y, const size_t x) {
    return (nc * height + y) * width + x;
}

__global__ void downscale(float* gradInput_data, const float* gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    // Utilize 1D indexing for simplicity
    long ii = blockIdx.x * blockDim.x + threadIdx.x + gridDim.x * blockDim.x * (blockIdx.y * blockDim.y + threadIdx.y);
    if (ii >= no_elements) return;

    int ipidx_base = translate_idx_inv(ii, d1, d2, d3, scale_factor, 0, 0);
    for (int i = 0; i < scale_factor; ++i) {
        for (int j = 0; j < scale_factor; ++j) {
            // Calculate ipidx using a base offset
            int ipidx = ipidx_base + d3 * j + i;
            gradInput_data[ii] += gradOutput_data[ipidx];
        }
    }
}