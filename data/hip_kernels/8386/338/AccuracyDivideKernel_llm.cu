#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel to compute accuracy directly in device
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use a single thread to avoid race conditions and unnecessary computation
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}