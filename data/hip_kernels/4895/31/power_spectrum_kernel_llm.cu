#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    // Use shared memory for performance improvement
    extern __shared__ float shared_A_in[];

    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    
    // Load input data into shared memory
    for (int idx = thread_id; idx < row_length; idx += blockDim.x) {
        shared_A_in[idx] = Ar[idx];
    }
    __syncthreads(); // Ensure all data is loaded before computation

    for (int idx = thread_id + 1; idx < half_length; idx += blockDim.x) { // Start loop from idx = 1

        float2 val = reinterpret_cast<float2 *>(shared_A_in)[idx]; // Use shared memory
        float ret = val.x * val.x + val.y * val.y;

        if (use_power) {
            Aw[idx] = ret;
        } else {
            Aw[idx] = sqrtf(ret);
        }
    }

    // Handle special case with thread 0
    if (thread_id == 0) {
        float real = shared_A_in[0];
        float im = shared_A_in[row_length];

        if (use_power) {
            Aw[0] = real * real;
            Aw[half_length] = im * im;
        } else {
            Aw[0] = fabs(real);
            Aw[half_length] = fabs(im);
        }
    }
}