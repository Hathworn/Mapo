#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Calculate row index using block and thread identifiers
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    // Calculate column index using block and thread identifiers
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check index bounds only once for better efficiency
    if (i < num_rows && j < num_cols) {
        float f = feats[i * ldf + j];
        feats_sq[i * lds + j] = f * f;
    }
}