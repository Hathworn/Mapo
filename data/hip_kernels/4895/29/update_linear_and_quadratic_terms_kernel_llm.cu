#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_linear_and_quadratic_terms_kernel( int32_t n, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    // Cache cur_weight in a register
    float cur_weight = *cur_tot_weight;
    float val = 1.0f;

    // Optimize computation by avoiding casts
    if (max_count > 0) {
        float new_scale = max(cur_weight, (float)max_count) / max_count;
        float prior_scale_change = new_scale - 1.0f;
        val += prior_scale_change;
    }

    int32_t total_threads = blockDim.x * gridDim.x;
    for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += total_threads) {
        // Calculate diag_idx using register variables to reduce recomputation
        int32_t diag_idx = i * (i + 3) / 2; // Simplified calculation of triangular index
        quadratic[diag_idx] += val;
    }

    // Use a single thread to update the shared linear term
    if (threadIdx.x == 0) {
        atomicAdd(&linear[0], val * prior_offset);
    }
}