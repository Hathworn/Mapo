#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void check_if_unique(const unsigned *keys, unsigned *is_unique, size_t kSize) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize first element uniquely
    if (id == 0) {
        is_unique[0] = 1;
    } 
    // Check bounds and uniqueness for other elements
    else if (id < kSize) {
        is_unique[id] = (keys[id] != keys[id - 1]);
    }
}