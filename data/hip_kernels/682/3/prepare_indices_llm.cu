#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prepare_indices(const unsigned num_keys, unsigned *data) {
    // Calculate the 1D global thread index more efficiently
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num_keys) {
        data[index] = index; // Assign index directly to the data array
    }
}