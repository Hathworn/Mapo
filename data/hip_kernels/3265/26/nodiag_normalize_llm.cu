#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void nodiag_normalize(double *A, double *I, int n, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Use a single combined condition to reduce nesting
    if (x < n && y < n && x == i && x != y) {
        double diagVal = A[i*n + i]; // Cache the diagonal value for reuse
        I[x*n + y] /= diagVal;
        A[x*n + y] /= diagVal;
    }
}