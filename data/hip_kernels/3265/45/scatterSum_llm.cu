#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scatterSum(int N, float *input, float *output) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= N) return;

    // No need for a temporary variable 'a'
    // Simplify atomic operation within loop
    for(int j = 0; j < N; ++j) {
        int index = (j + i) % N;
        atomicAdd(&output[index], input[i]);
    }
    return;
}