#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void optimizedKernel(int N, double *input, double *output) {
    int global_i = blockIdx.x * blockDim.x + threadIdx.x;
    if(global_i < N) {
        double sum = 0.0;

        // Use a loop unrolling technique to enhance memory access and performance
        for (int i = 0; i < N; i += 4) {
            if (i < N) sum += input[i];
            if (i + 1 < N) sum += input[i + 1];
            if (i + 2 < N) sum += input[i + 2];
            if (i + 3 < N) sum += input[i + 3];
        }
        
        // Calculate the average
        output[global_i] = sum / N;
    }
    return;
}