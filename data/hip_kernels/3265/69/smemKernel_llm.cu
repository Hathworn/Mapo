#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smemKernel(int N, double *input, double *output) {
    int b_size = blockDim.x, b_idx = blockIdx.x, t_idx = threadIdx.x;
    int global_i = b_size * b_idx + t_idx, n_chk = (N + SHARE_SIZE - 1) / SHARE_SIZE;
    __shared__ double buff[SHARE_SIZE];
    
    double sum = 0.0; // Accumulate partial sums to avoid shared memory contention

    for(int q = 0; q < n_chk; ++q) {
        int left = q * SHARE_SIZE, right = min(left + SHARE_SIZE, N);
        
        for(int i = t_idx + left; i < right; i += b_size) {
            buff[i - left] = input[i];
        }

        __syncthreads();

        // Use local accumulator instead of updating global memory directly in loop
        for(int i = left; i < right; ++i) {
            sum += buff[i - left];
        }

        __syncthreads();
    }

    output[global_i] = sum / N; // Divide after accumulation to reduce division operations
}