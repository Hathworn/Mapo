#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaussjordan(double *A, double *I, int n, int i)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure indices are within bounds
    if (x < n && y < n) {
        // Use shared memory to reduce global memory access 
        __shared__ double shared_A_i[1024]; // or adjust size to fit n per block constraints
        __shared__ double shared_I_i[1024];

        // Load a row of A and I to shared memory once
        if (y == 0) {
            shared_A_i[threadIdx.x] = A[i * n + x];
            shared_I_i[threadIdx.x] = I[i * n + x];
        }
        __syncthreads();

        if (x != i) {
            I[x * n + y] -= shared_I_i[threadIdx.x] * A[x * n + i];
            if (y != i) {
                A[x * n + y] -= shared_A_i[threadIdx.x] * A[x * n + i];
            }
        }
    }
}