#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaGetError(int N, double *ana, double *cur, double *e_sum) {
    // Use shared memory to reduce global memory access
    __shared__ double sharedSum[256]; // Assuming blockDim.x is at most 256
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    double diff = 0.0;
    if (index < (N+1)*(N+1)) {
        diff = (ana[index] - cur[index]) * (ana[index] - cur[index]);
    }
    sharedSum[threadIdx.x] = diff;

    // Perform reduction in shared memory
    __syncthreads();
    if (threadIdx.x < 128) sharedSum[threadIdx.x] += sharedSum[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64) sharedSum[threadIdx.x] += sharedSum[threadIdx.x + 64];
    __syncthreads();
    if (threadIdx.x < 32) {
        sharedSum[threadIdx.x] += sharedSum[threadIdx.x + 32];
        sharedSum[threadIdx.x] += sharedSum[threadIdx.x + 16];
        sharedSum[threadIdx.x] += sharedSum[threadIdx.x + 8];
        sharedSum[threadIdx.x] += sharedSum[threadIdx.x + 4];
        sharedSum[threadIdx.x] += sharedSum[threadIdx.x + 2];
        sharedSum[threadIdx.x] += sharedSum[threadIdx.x + 1];
    }

    // Sum up results from each block
    if (threadIdx.x == 0) atomicAdd(e_sum, sharedSum[0]);
}