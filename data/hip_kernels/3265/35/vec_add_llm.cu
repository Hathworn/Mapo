#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_add(int N, int *A, int *B, int *C) {
    // Utilize global index directly
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check boundary within index calculation itself
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}