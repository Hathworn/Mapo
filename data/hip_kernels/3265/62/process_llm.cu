#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float dothings(int t, int sz, float *input) {
    float ans = 0;
    for (int i = 0; i < 12; ++i) {
        ans += input[(i + t) % sz];
    }
    return ans;
}

__global__ void process(int N_step, int N_inst, float *input, float *output) {
    int g_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (g_id >= N_inst) return;

    float ans = 0.;
    // Coalesce global memory access by loading data into shared memory
    extern __shared__ float local_data[];
    #pragma unroll
    for (int i = 0; i < VEC_SIZE; ++i) {
        local_data[i] = input[VEC_SIZE * g_id + i];
    }
    __syncthreads();

    #pragma unroll
    for (int t = 0; t < N_step; ++t) {
        ans += dothings(t, VEC_SIZE, local_data);
    }
    
    output[g_id] = ans;
}
```
