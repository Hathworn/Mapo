#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diag_normalize(double *A, double *I, int n, int i) {
    // Calculate unique thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Only one thread normalizes the diagonal element at (i, i)
    if (idx == i) {
        I[i * n + i] /= A[i * n + i];
        A[i * n + i] /= A[i * n + i];
    }
}