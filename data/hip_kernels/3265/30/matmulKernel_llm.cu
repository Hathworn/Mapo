#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmulKernel(float *A, float *B, float *C, int rA, int cA, int cB) {
    // Optimize by reusing loaded C value and using registers
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < rA && j < cB) {
        float sum = 0.0f; // Use register for accumulation
        for (int k = 0; k < cA; ++k) {
            sum += A[i * cA + k] * B[k * cB + j];
        }
        C[i * cB + j] = sum; // Store the result back to C
    }
}