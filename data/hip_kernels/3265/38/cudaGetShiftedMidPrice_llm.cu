#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaGetShiftedMidPrice(int N_inst, int batch_size, float *alphas, float *mid, float *shifted_prc) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Linear index for threads
    int stride = blockDim.x * gridDim.x;              // Total number of threads
    if (blockIdx.x < N_inst) {
        for (int i = idx; i < batch_size; i += stride) {
            shifted_prc[blockIdx.x * batch_size + i] = (1.0f + alphas[blockIdx.x * batch_size + i]) * mid[i];
        }
    }
}