#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smemKernel(int N, float *input, float *output){
    int b_size = blockDim.x, b_idx = blockIdx.x, t_idx = threadIdx.x;
    int global_i = b_size * b_idx + t_idx;
    __shared__ float buff[SHARE_SIZE];
    
    // Initialize output element
    float sum = 0.0f; 

    for(int q = 0, stride = SHARE_SIZE; q < N; q += stride){
        int left = q;
        int right = min(left + SHARE_SIZE, N);
        
        // Load input to shared memory in a coalesced manner
        if (t_idx + left < right) {
            buff[t_idx] = input[t_idx + left];
        }
        __syncthreads();
        
        // Accumulate values from shared memory to local sum
        for(int i = t_idx; i < (right - left); i += b_size) {
            sum += buff[i];
        }
        __syncthreads();
    }
    
    // Store the average in output
    if(global_i < N) {
        output[global_i] = sum / N;
    }
}