#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gatherSum(int N, float *input, float *output) {
    // Calculate thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Return if thread index is out of bounds
    if (i >= N) return;

    // Initialize sum to zero
    float sum = 0.0f;
    
    // Accumulate input values in shared memory
    for (int j = 0; j < N; ++j) {
        sum += input[j];
    }
    
    // Write result to output
    output[i] = sum;
}