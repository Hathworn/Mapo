#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void process(int N_step, int N_inst, float *input, float *output) {
    int g_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (g_id >= N_inst) return;

    float ans = 0.0f;
    // Use shared memory to reduce global memory access
    extern __shared__ float shared_input[];
    
    // Load data into shared memory
    for (int i = threadIdx.x; i < VEC_SIZE; i += blockDim.x) {
        shared_input[i] = input[i + VEC_SIZE * g_id];
    }
    __syncthreads();

    // Accumulate the values in the shared memory
    for (int t = 0; t < N_step; ++t) {
        for (int i = 0; i < 12; ++i) {
            ans += shared_input[(i + t) % VEC_SIZE];
        }
    }

    output[g_id] = ans;
}