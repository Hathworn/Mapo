#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scatterSum(int N, float *input, float *output){
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread index is within bounds
    if(i < N) {
        // Unroll the loop for potential optimization
        for(int j=0; j<N; j+=4){
            // Atomic operations for updating output to prevent race conditions
            atomicAdd(output+j, input[i]);
            if (j+1 < N) atomicAdd(output+j+1, input[i]);
            if (j+2 < N) atomicAdd(output+j+2, input[i]);
            if (j+3 < N) atomicAdd(output+j+3, input[i]);
        }
    }
}