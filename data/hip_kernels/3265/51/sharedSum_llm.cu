#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sharedSum(int N, float *input, float *output) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Use shared memory for intra-block summation
    __shared__ float tmp[BLOCK_SIZE];
    
    // Initialize shared memory for the current thread
    tmp[threadIdx.x] = 0.0f;

    // Load data into register
    float a = input[i];

    // Combine atomic operation with thread synchronization
    atomicAdd(&tmp[threadIdx.x], a);
    __syncthreads();

    // Write the result from shared memory to output
    output[i] = tmp[threadIdx.x];
}