#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved global kernel function for vector addition
__global__ void vec_add(int N, int *A, int *B, int *C) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // Use valid index to perform element-wise vector multiplication
    if (i < N) {
        C[i] = A[i] * B[i];
    }
}