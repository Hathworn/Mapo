#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void optimizedKernel(int N, float *input, float *output) {
    // Use shared memory to prevent loading from global memory repeatedly
    extern __shared__ float shared_input[];
    int global_i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Only load necessary data into shared memory once
    if (global_i < N) {
        if (threadIdx.x < N) {
            shared_input[threadIdx.x] = input[threadIdx.x];
        }
        __syncthreads();

        // Compute the output using shared memory data
        float sum = 0.0f;
        for(int i = 0; i < N; ++i) {
            sum += shared_input[i];
        }
        output[global_i] = sum / N;
    }
}