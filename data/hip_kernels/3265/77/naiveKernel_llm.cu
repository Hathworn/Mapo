#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void optimizedKernel(int N, float *input, float *output) {
    // Use shared memory for better memory access efficiency
    extern __shared__ float sharedData[];
    int tid = threadIdx.x;
    int blockSize = blockDim.x;

    // Initialize shared memory
    float localSum = 0.0f;
    for (int i = tid; i < N; i += blockSize) {
        localSum += input[i];
    }
    sharedData[tid] = localSum;
    __syncthreads();

    // Parallel reduction to compute the sum using shared memory
    for (int stride = blockSize / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedData[tid] += sharedData[tid + stride];
        }
        __syncthreads();
    }

    // Normalize the result and store in output
    if (tid == 0) {
        *output = sharedData[0] / N;
    }
}