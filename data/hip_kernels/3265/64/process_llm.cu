#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void process(int N_step, int N_inst, float *input, float *output) {
    int b_id = blockIdx.x, t_id = threadIdx.x;
    if (b_id >= N_inst) return;
    __shared__ float ans;
    float val = 0.0f;

    // Initialize shared memory once before loop
    if (!t_id) ans = 0.0f;
    __syncthreads();
    
    // Load input into register to reduce global memory access
    if (t_id < VEC_SIZE) val = input[VEC_SIZE * b_id + t_id];
    __syncthreads();

    for (int t = 0; t < N_step; ++t) {
        int start = t % VEC_SIZE;

        // Use a single atomic operation for each thread if within bounds
        if (t_id < VEC_SIZE && (t_id >= start && t_id < start + 12)) {
            atomicAdd(&ans, val);
        }
        __syncthreads();
    }

    // Write to global memory once outside the loop
    if (!t_id) output[b_id] = ans;
    return;
}
```
