#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_zero(double *A, double *I, int n, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check only once if within bounds, reducing nested ifs
    if (x < n && y < n && x != i && y == i) {
        A[x * n + y] = 0;
    }
}