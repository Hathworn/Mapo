#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smemKernel(int N, float *input, float *output) {
    int b_size = blockDim.x, b_idx = blockIdx.x, t_idx = threadIdx.x;
    int global_i = b_size * b_idx + t_idx;
    int n_chk = (N + SHARE_SIZE - 1) / SHARE_SIZE;
    __shared__ float buff[SHARE_SIZE];
    
    float temp_sum = 0.0f; // Accumulator for partial sum

    for (int q = 0; q < n_chk; ++q) {
        int left = q * SHARE_SIZE, right = min(left + SHARE_SIZE, N);
        
        // Efficiently load data into shared memory
        for (int i = t_idx + left; i < right; i += b_size) {
            buff[i - left] = input[i];
        }
        __syncthreads();
        
        if (global_i < N) {
            // Accumulate the local sum from shared memory
            for (int i = left; i < right; ++i) {
                temp_sum += buff[i - left];
            }
        }
        __syncthreads();
    }
    
    if (global_i < N) {
        // Store the averaged result in the output array
        output[global_i] = temp_sum / N;
    }
}