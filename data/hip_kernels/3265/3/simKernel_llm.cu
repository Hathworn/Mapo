#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simKernel(int N_stgy, int N_batch, double *alpha, double *mid, double *gap, int *late, int *pos, int *rest_lag, double *prof, double *last_prc, int *cnt, double fee) {
    int global_i = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_i >= N_stgy) return;

    int start = global_i * N_batch + rest_lag[global_i];
    int end = global_i * N_batch + N_batch;
    
    // Loop optimization: reduce redundant calculations
    for (int i = start; i < end; ++i) {
        double alpha_mid = alpha[i] * mid[i % N_batch];
        double gap_fee = gap[i % N_batch] + fee;

        if (alpha_mid > gap_fee || alpha_mid < -gap_fee) {
            double mid_value = mid[i % N_batch];
            double gap_value = gap[i % N_batch];
            
            if (alpha_mid > gap_fee && pos[global_i] < 1) {
                last_prc[global_i] = mid_value + gap_value + fee;
                prof[global_i] -= (1 - pos[global_i]) * last_prc[global_i];
                cnt[global_i] += 1 - pos[global_i];
                pos[global_i] = 1;
                i += late[i % N_batch];
            } else if (alpha_mid < -gap_fee && pos[global_i] > -1) {
                last_prc[global_i] = mid_value - gap_value - fee;
                prof[global_i] += (pos[global_i] + 1) * last_prc[global_i];
                cnt[global_i] += pos[global_i] + 1;
                pos[global_i] = -1;
                i += late[i % N_batch];
            }
        }
    }
    rest_lag[global_i] = end - (global_i * N_batch); // Ensures proper reset of rest_lag
}