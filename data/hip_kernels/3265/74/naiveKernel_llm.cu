#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void optimizedKernel(int N, float *input, float *output) {
    // Calculate global index
    int global_i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if within bounds
    if (global_i < N) {
        // Improvements: use shared memory to reduce global memory access
        extern __shared__ float shared_input[];
        
        // Cooperative loading into shared memory
        for (int i = threadIdx.x; i < N; i += blockDim.x) {
            shared_input[i] = input[i];
        }
        __syncthreads();
        
        // Accumulate results using shared memory
        float sum = 0.0f;
        for (int i = 0; i < N; ++i) {
            sum += shared_input[i];
        }

        // Store the result in global memory
        output[global_i] = sum / N;
    }
}