#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void naiveKernel(int N, float *input, float *output) {
    // Use shared memory for better memory access speed
    __shared__ float sharedInput[1024];  // Example size, should match with the block size
    int global_i = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_i = threadIdx.x;

    // Load data into shared memory
    if (thread_i < N) {
        sharedInput[thread_i] = input[thread_i];
    }
    __syncthreads();

    if (global_i < N) {
        float sum = 0.0f;
        // Use shared memory for sum calculation
        for (int i = 0; i < N; ++i) {
            sum += sharedInput[i];
        }
        output[global_i] = sum / N;
    }
}