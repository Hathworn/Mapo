#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simKernel(int N_stgy, int N_batch, float *alpha, float *mid, float *gap, int *late, int *pos, int *rest_lag, float *prof, float *last_prc, int *cnt, float fee) {
    int global_i = blockIdx.x * blockDim.x + threadIdx.x;
    if (global_i >= N_stgy) return;

    int start = global_i * N_batch + rest_lag[global_i];
    int end = global_i * N_batch + N_batch;
    int i;

    // Cache repeated calculations for batch indices
    for (i = start; i < end; ++i) {
        int batchIndex = i % N_batch;
        float currentAlphaMid = alpha[i] * mid[batchIndex];
        float currentGap = gap[batchIndex];

        // Optimize condition checks and arithmetic calculations
        if (currentAlphaMid > currentGap + fee || currentAlphaMid < -currentGap - fee) {
            if (currentAlphaMid > currentGap + fee && pos[global_i] < 1) {
                last_prc[global_i] = mid[batchIndex] + currentGap + fee;
                prof[global_i] -= (1 - pos[global_i]) * last_prc[global_i];
                cnt[global_i] += 1 - pos[global_i];
                pos[global_i] = 1;
                i += late[batchIndex];
            } else if (currentAlphaMid < -currentGap - fee && pos[global_i] > -1) {
                last_prc[global_i] = mid[batchIndex] - currentGap - fee;
                prof[global_i] += (pos[global_i] + 1) * last_prc[global_i];
                cnt[global_i] += pos[global_i] + 1;
                pos[global_i] = -1;
                i += late[batchIndex];
            }
        }
    }
    rest_lag[global_i] = i - end;
}