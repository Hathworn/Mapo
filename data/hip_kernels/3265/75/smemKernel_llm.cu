#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smemKernel(int N, float *input, float *output) {
    int b_size = blockDim.x, b_idx = blockIdx.x, t_idx = threadIdx.x;
    int global_i = b_size * b_idx + t_idx;
    __shared__ float buff[SHARE_SIZE];
    
    // Reset output, only once instead of at every iteration
    if(global_i < N) {
        output[global_i] = 0.0f;
    }

    int n_chk = (N + SHARE_SIZE - 1) / SHARE_SIZE;
    for (int q = 0; q < n_chk; ++q) {
        int left = q * SHARE_SIZE, right = min(left + SHARE_SIZE, N);
        
        // Load elements into shared memory
        if (t_idx + left < right) {
            buff[t_idx] = input[t_idx + left];
        }
        __syncthreads();
        
        // Accumulate results
        if (global_i < N) {
            for (int i = 0; i < right - left; ++i) {
                output[global_i] += buff[i];
            }
        }
        __syncthreads();
    }
    
    // Final result normalization
    if(global_i < N) {
        output[global_i] /= N;
    }
}