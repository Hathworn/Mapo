#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/* This file is copied from https://github.com/jzbonter/mc-cnn */
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void rho(float *x, int size, float lambda)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if within bounds and compute result
    if (id < size) {
        // Prefetch value to register for reuse
        float value = x[id];
        x[id] = 1 - exp(-value / lambda);
    }
}