#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TB 128
#define DISP_MAX 256

__global__ void Normalize_get_norm_(float *input, float *norm, int size1, int size23, int size023)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size023) {
        int dim23 = id % size23;
        int dim0 = id / size23;

        float sum = 0.0;

        // Use pointers to reduce complex indexing in the loop
        float* input_ptr = input + dim0 * size1 * size23 + dim23;
        for (int dim1 = 0; dim1 < size1; dim1++) {
            float x = *input_ptr;
            sum += x * x;
            input_ptr += size23; // Move to the next element in the same column
        }
        norm[dim0 * size23 + dim23] = sum + 1e-5;
    }
}