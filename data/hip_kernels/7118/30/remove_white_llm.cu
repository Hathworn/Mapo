#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/* This file is copied from https://github.com/jzbonter/mc-cnn */
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void remove_white(float *x, float *y, int size)
{
    // Calculate unique global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure id is within bounds to prevent accessing out of bounds memory
    if (id >= size) return;

    // Replace white color value (255) with zero
    if (x[id] == 255.0f) {
        y[id] = 0.0f;
    }
}