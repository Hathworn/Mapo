#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TB 128
#define DISP_MAX 256

__global__ void copy_fill(float *in, float *out, int size, int in_size2, int in_size3, int out_size2, int out_size3)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (id >= size) return; // Exit early if thread ID is out of bounds

    int out_x = id % out_size3;
    int out_y = id / out_size3;

    int in_x = out_x - (out_size3 - in_size3) / 2;
    int in_y = out_y - (out_size2 - in_size2) / 2;

    // Clamp coordinates to avoid branch divergence
    in_x = max(0, min(in_size3 - 1, in_x));
    in_y = max(0, min(in_size2 - 1, in_y));

    out[id] = in[in_y * in_size3 + in_x];
}