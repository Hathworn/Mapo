#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TB 128
#define DISP_MAX 256

__global__ void subpixel_enchancement(float *d0, float *c2, float *out, int size, int dim23, int disp_max) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if id is within bounds
    if (id >= size) return;

    int d = d0[id];
    out[id] = d;

    // Ensure d is within valid range for subpixel enhancement calculation
    if (1 <= d && d < disp_max - 1) {
        float cn = c2[(d - 1) * dim23 + id];
        float cz = c2[d * dim23 + id];
        float cp = c2[(d + 1) * dim23 + id];
        
        // Calculate denominator and adjust with threshold check
        float denom = 2 * (cp + cn - 2 * cz);
        if (denom > 1e-5) {
            float adjustment = (cp - cn) / denom;
            
            // Clamp adjustment value and update output
            out[id] = d - fminf(1.0f, fmaxf(-1.0f, adjustment));
        }
    }
}