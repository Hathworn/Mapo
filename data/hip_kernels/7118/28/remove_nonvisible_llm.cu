#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/* This file is copied from https://github.com/jzbonter/mc-cnn */
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void remove_nonvisible(float *y, int size, int size3)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Exit early if id is out of bounds
    if (id >= size) return;

    int x = id % size3; // Compute x once and reuse
    if (y[id] >= x) {
        y[id] = 0;
    }
}