#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rectify(unsigned char* image, unsigned height, unsigned width, int thread_count)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Determine total number of pixels
    int total_pixels = height * width * 4;
    // Calculate stride based on total thread count
    int stride = blockDim.x * gridDim.x;

    // Loop through assigned pixels with striding
    for (int j = idx; j < total_pixels; j += stride) 
    {
        if (image[j] < 127) image[j] = 127;
    }
}