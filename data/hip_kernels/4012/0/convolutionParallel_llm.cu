#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolutionParallel(unsigned char* image, unsigned char* new_image, unsigned height, unsigned width, int thread_count, int convolution_size)
{
    // Calculate pixel offset
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int width_out = width - convolution_size + 1;
    int height_out = height - convolution_size + 1;

    // Loop over pixels of smaller image
    for (int i = offset; i < width_out * height_out * 4; i += thread_count)
    {
        int row = i / (4 * width_out);
        int col = i % (4 * width_out);
        int reference_pixel_offset = 4 * row * width + col;
        float sum = 0.0f;
        const float *w;

        // Select weights based on convolution size
        if (convolution_size == 3)
        {
            __shared__ const float w_shared[9];
            if (threadIdx.x < 9) {
                w_shared[threadIdx.x] = ((const float[9]){1, 2, -1, 2, 0.25, -2, 1, -2, -1})[threadIdx.x];
            }
            __syncthreads();
            w = w_shared;
        }
        else if (convolution_size == 5)
        {
            __shared__ const float w_shared[25];
            if (threadIdx.x < 25) {
                w_shared[threadIdx.x] = ((const float[25]){0.5, 0.75, 1, -0.75, -0.5, 0.75, 1, 2, -1, -0.75, 1, 2, 0.25, -2, -1, 0.75, 1, -2, -1, -0.75, 0.5, 0.75, -1, -0.75, -0.5})[threadIdx.x];
            }
            __syncthreads();
            w = w_shared;
        }
        else if (convolution_size == 7)
        {
            __shared__ const float w_shared[49];
            if (threadIdx.x < 49) {
                w_shared[threadIdx.x] = ((const float[49]){0.25, 0.3, 0.5, 0.75, -0.5, -0.3, -0.25, 0.3, 0.5, 0.75, 1, -0.75, -0.5, -0.3, 0.5, 0.75, 1, 2, -1, -0.75, -0.5, 0.75, 1, 2, 0.25, -2, -1, -0.75, 0.5, 0.75, 1, -2, -1, -0.75, -0.5, 0.3, 0.5, 0.75, -1, -0.75, -0.5, -0.3, 0.25, 0.3, 0.5, -0.75, -0.5, -0.3, -0.25})[threadIdx.x];
            }
            __syncthreads();
            w = w_shared;
        }

        // Apply convolution filter
        for (int j = 0; j < convolution_size; j++)
        {
            for (int k = 0; k < convolution_size; k++)
            {
                sum += image[reference_pixel_offset + 4 * k + 4 * j * width] * w[j * convolution_size + k];
            }
        }

        // Clamp and apply alpha channel setting
        sum = fminf(fmaxf(sum, 0.0f), 255.0f);
        if ((i + 1) % 4 == 0) sum = 255.0f;

        new_image[i] = (int)sum;
    }
}