#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pool(unsigned char* image, unsigned char* new_image, unsigned height, unsigned width, int thread_count) {
    // Process image with shared memory optimization
    extern __shared__ unsigned char sharedImage[];

    int offset = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    int stride = thread_count * 4;

    // Load a tile of the image into shared memory
    for (int i = offset; i < min(offset + stride, width * height * 4); i += stride) {
        sharedImage[threadIdx.x + (i - offset)] = image[i];
    }
    __syncthreads();

    for (int i = offset; i < (width * height); i += stride) {
        int x = i % (width * 2) * 2;
        int y = i / (width * 2);
        int p1 = 8 * width * y + x;
        int p2 = 8 * width * y + x + 4;
        int p3 = 8 * width * y + x + 4 * width;
        int p4 = 8 * width * y + x + 4 * width + 4;

        unsigned r[] = { sharedImage[p1], sharedImage[p2], sharedImage[p3], sharedImage[p4] };
        unsigned g[] = { sharedImage[p1+1], sharedImage[p2+1], sharedImage[p3+1], sharedImage[p4+1] };
        unsigned b[] = { sharedImage[p1+2], sharedImage[p2+2], sharedImage[p3+2], sharedImage[p4+2] };
        unsigned a[] = { sharedImage[p1+3], sharedImage[p2+3], sharedImage[p3+3], sharedImage[p4+3] };

        unsigned rMax = max(max(r[0], r[1]), max(r[2], r[3]));
        unsigned gMax = max(max(g[0], g[1]), max(g[2], g[3]));
        unsigned bMax = max(max(b[0], b[1]), max(b[2], b[3]));
        unsigned aMax = max(max(a[0], a[1]), max(a[2], a[3]));

        new_image[i] = rMax;
        new_image[i+1] = gMax;
        new_image[i+2] = bMax;
        new_image[i+3] = aMax;
    }
}