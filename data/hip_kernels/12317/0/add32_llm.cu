#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void add32(float* A, float *B, int size) {
    // Use simple 1D grid-stride loop for better performance and easier readability
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < size; i += stride) {
        A[i] = A[i] + B[i];
    }
}