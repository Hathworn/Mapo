#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Module to test HIP module loading and execution.
*/

#ifdef __cplusplus
extern "C" {
#endif

/// Sets the first N elements of array to value.
#ifdef __cplusplus
}
#endif
__global__ void testMemset(float* array, float value, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation
    if(i < N){
        array[i] = value;
    }
}