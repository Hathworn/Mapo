#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void Ring2_kernel( float *A, float *BP, int *corrAB, int *mask, int *m, int ring, int c, int h, int w )
{
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;

    if (id1 < size && mask[id1] != 0) {
        int y2 = corrAB[2 * id1 + 1], x2 = corrAB[2 * id1];
        int start_x = MAX(0, x2 - ring), end_x = MIN(w - 1, x2 + ring);
        int start_y = MAX(0, y2 - ring), end_y = MIN(h - 1, y2 + ring);

        // Unroll the inner loops for better performance
        for (int _x2 = start_x; _x2 <= end_x; ++_x2) {
            for (int _y2 = start_y; _y2 <= end_y; ++_y2) {
                m[_y2 * w + _x2] = 1;
            }
        }
    }
}
