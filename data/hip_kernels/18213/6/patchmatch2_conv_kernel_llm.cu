#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void patchmatch2_conv_kernel( float *A, float *B, float *AP, float *BP, float *conv, int *prev_corrAB_upsampled, int patch, int s_rad, int c, int h, int w )
{
    int h1 = h, w1 = w;
    int _id = blockIdx.x * blockDim.x + threadIdx.x;
    int size1 = h * w;
    int s_size = 2 * s_rad + 1;
    int s_n = s_size * s_size;

    // Check within bounds
    if (_id >= size1 * s_n) return;
  
    conv[_id] = -1;

    int id1 = _id / s_n, s_idx = _id % s_n;
    int y1 = id1 / w1, x1 = id1 % w1;
    int dy2 = s_idx / s_size - s_rad, dx2 = s_idx % s_size - s_rad;

    int x2 = prev_corrAB_upsampled[2 * id1 + 0];
    int y2 = prev_corrAB_upsampled[2 * id1 + 1];

    int new_y2 = y2 + dy2;
    int new_x2 = x2 + dx2;

    // Boundary check
    if (new_x2 < 0 || new_x2 >= w1 || new_y2 < 0 || new_y2 >= h1) return;

    int kernel_radius = (patch - 1) / 2;
    float conv_result = 0.0f;
    int cnt = 0;

    for (int dy = -kernel_radius; dy <= kernel_radius; dy++) {
        for (int dx = -kernel_radius; dx <= kernel_radius; dx++) {
            int xx1 = x1 + dx, yy1 = y1 + dy;
            int xx2 = new_x2 + dx, yy2 = new_y2 + dy;
            if (xx1 >= 0 && xx1 < w1 && yy1 >= 0 && yy1 < h1 &&
                xx2 >= 0 && xx2 < w1 && yy2 >= 0 && yy2 < h1)
            {
                int _id1 = yy1 * w1 + xx1, _id2 = yy2 * w1 + xx2;
                
                // Loop unrolling and memory coalescing
                #pragma unroll
                for (int dc = 0; dc < c; dc++) {
                    float term1A = A[dc * size1 + _id1];
                    float term1B = B[dc * size1 + _id2];
                    conv_result += term1A * term1B;

                    float term2A = AP[dc * size1 + _id1];
                    float term2B = BP[dc * size1 + _id2];
                    conv_result += term2A * term2B;
                }
                cnt++;
            }
        }
    }

    // Avoid division by zero
    if (cnt > 0)
        conv[_id] = conv_result / static_cast<float>(cnt);
}