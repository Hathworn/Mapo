#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void histogram_kernel(float *I, float *minI, float *maxI, float *mask, int nbins, int c, int h, int w, float *hist)
{
    // Calculate the global thread ID
    int _id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;
    
    // Check if the thread ID is within bounds
    if (_id < c * size) {
        int id = _id % size;
        int dc = _id / size;
        
        // Skip mask elements below the threshold
        if (mask[id] >= EPS) {
            float val = I[_id];
            float _minI = minI[dc];
            float _maxI = maxI[dc];
            
            // Adjust min and max to avoid division by zero
            if (_minI == _maxI) {
                _minI -= 1.0f;
                _maxI += 1.0f;
            }
            
            // Only process values within the range
            if (_minI <= val && val <= _maxI) {
                // Compute and clamp bin index
                int idx = MIN(static_cast<int>((val - _minI) / (_maxI - _minI) * nbins), nbins - 1);
                int index = dc * nbins + idx;
                
                // Use atomic add to update the histogram
                atomicAdd(&hist[index], 1.0f);
            }
        }
    }
}