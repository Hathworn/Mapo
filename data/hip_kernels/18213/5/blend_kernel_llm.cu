#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void blend_kernel(float *A, float *BP, float *M, float *AP, float alpha, int c, int h, int w)
{
    int _id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;

    // Use shared memory to improve memory access efficiency
    __shared__ float shared_M[TB];
    
    if (_id < size) {
        shared_M[threadIdx.x] = M[_id];
    }
    __syncthreads();

    if (_id < c * size) {
        int id = _id % size, dc = _id / size;
        
        // Cache M[id] read from shared memory
        float current_M = shared_M[id % TB];
        float weight = current_M < 0.05f ? 0.f : alpha;

        AP[_id] = A[_id] * weight + BP[_id] * (1.f - weight);
    }
}