#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void hist_remap2_kernel(float *I, int nI, float *mI, float *histJ, float *cumJ, float *_minJ, float *_maxJ, int nbins, float *_sortI, int *_idxI, float *R, int c, int h, int w) {
    int _id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;

    if (_id >= c * size) return;

    // Precompute values used multiple times
    int id = _id % size, dc = _id / size;
    float minJ = _minJ[dc];
    float maxJ = _maxJ[dc];
    float stepJ = (maxJ - minJ) / nbins;
    int idxI = _idxI[_id] - 1;

    // Immediate return if condition not met
    if (mI[idxI] < EPS) return;

    int offset = h * w - nI;
    int cdf = id - offset;
    int s = 0, e = nbins - 1, binIdx = -1;
    float cdf_e, cdf_s;

    // Binary search for finding binIdx
    while (s <= e) {
        int m = (s + e) / 2;
        cdf_e = (m == nbins - 1) ? cumJ[dc * nbins + m] + 0.5f : cumJ[dc * nbins + m];
        cdf_s = (m == 0) ? -0.5f : cumJ[dc * nbins + m - 1];

        if (cdf >= cdf_e) {
            s = m + 1;
        } else if (cdf < cdf_s) {
            e = m - 1;
        } else {
            binIdx = m;
            break;
        }
    }

    // Final calculation
    float hist = histJ[dc * nbins + binIdx];
    cdf_e = cumJ[dc * nbins + binIdx];
    cdf_s = cdf_e - hist;
    float ratio = MIN(MAX((cdf - cdf_s) / (hist + 1e-8f), 0.0f), 1.0f);
    float activation = minJ + (static_cast<float>(binIdx) + ratio) * stepJ;

    R[dc * size + idxI] = activation;
}