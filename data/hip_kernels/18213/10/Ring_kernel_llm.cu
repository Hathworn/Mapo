#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void Ring_kernel(float *A, float *BP, int *corrAB, float *M, int ring, int c, int h, int w)
{
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;
    int ringSize = 2 * ring + 1;
    int ringPatch = ringSize * ringSize;

    if (id1 < size) {
        int y1 = id1 / w, x1 = id1 % w;
        int y2 = corrAB[2 * id1 + 1], x2 = corrAB[2 * id1 + 0];

        for (int dx = -ring; dx <= ring; dx++) {
            for (int dy = -ring; dy <= ring; dy++) {
                int pIdx = (dy + ring) * ringSize + (dx + ring);
                int _x2 = x2 + dx, _y2 = y2 + dy;

                // Optimize to reduce redundant computations
                if (_x2 >= 0 && _x2 < w && _y2 >= 0 && _y2 < h) {
                    int M_index_base = (y1 * w) * ringPatch + pIdx * w + x1;
                    int BP_index_base = _y2 * w + _x2;
                    for (int dc = 0; dc < c; dc++) {
                        int dc_size = dc * size;
                        M[dc_size + M_index_base] = BP[dc_size + BP_index_base];
                    }
                }
            }
        }
    }
    return;
}