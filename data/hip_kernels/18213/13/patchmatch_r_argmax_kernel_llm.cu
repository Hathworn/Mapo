#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void patchmatch_r_argmax_kernel(float *conv, float *target, float *match, int *correspondence, int c1, int h1, int w1, int h2, int w2) {
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    if (id1 >= h1 * w1) return; // Return early if out of bounds

    int size1 = h1 * w1, size2 = h2 * w2;
    double conv_max = -1e20;
    int best_x2 = 0, best_y2 = 0; // Store best x2 and y2 to reduce memory writes

    for (int id2 = 0; id2 < size2; id2++) { // Linearize y2 and x2 loop into single loop
        float conv_result = conv[id1 * size2 + id2]; // Direct access to conv element
        if (conv_result > conv_max) { // Check if this is the best result so far
            conv_max = conv_result; // Update max convolution value
            best_x2 = id2 % w2; // Calculate candidate x2
            best_y2 = id2 / w2; // Calculate candidate y2
        }
    }
    
    // Only update correspondence and matches once after determining best match
    correspondence[id1 * 2] = best_x2;
    correspondence[id1 * 2 + 1] = best_y2;
    int best_id2 = best_y2 * w2 + best_x2; // Calculate the best id2 once
    for (int c = 0; c < c1; c++) {
        match[c * size1 + id1] = target[c * size2 + best_id2]; // Update matches
    }
}