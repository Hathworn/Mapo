#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16

__global__ void MultiplyGPU(float* a, float* b, float* c, int t)
{
    // Calculate row and column indices
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    float aux = 0.0f;

    // Ensure indices are within bounds
    if (i < t && j < t)
    {
        // Use shared memory for improved cache efficiency
        __shared__ float a_shared[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float b_shared[BLOCK_SIZE][BLOCK_SIZE];

        for (int k = 0; k < t; k += BLOCK_SIZE)
        {
            // Load data into shared memory
            if (k + threadIdx.x < t && i < t)
                a_shared[threadIdx.y][threadIdx.x] = a[i * t + (k + threadIdx.x)];
            else
                a_shared[threadIdx.y][threadIdx.x] = 0.0f;

            if (k + threadIdx.y < t && j < t)
                b_shared[threadIdx.y][threadIdx.x] = b[(k + threadIdx.y) * t + j];
            else
                b_shared[threadIdx.y][threadIdx.x] = 0.0f;

            __syncthreads();

            // Perform partial multiplication
            for (int n = 0; n < BLOCK_SIZE; n++)
            {
                aux += a_shared[threadIdx.y][n] * b_shared[n][threadIdx.x];
            }

            __syncthreads();
        }

        // Write result to matrix c
        c[i * t + j] = aux;
    }
}