#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16

__device__ float* GetSubMatrix(float *a, int tam, int row, int col)
{
    return &a[tam * BLOCK_SIZE * row + BLOCK_SIZE * col];
}

__global__ void MultiplyGPUMult(float *a, float *b, float *c, int t)
{
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;
    int row = threadIdx.y;
    int col = threadIdx.x;

    float* Csub = GetSubMatrix(c, t, blockRow, blockCol);
    float Cvalue = 0.0f;

    for (int m = 0; m < t / BLOCK_SIZE; m++)
    {
        float* Asub = GetSubMatrix(a, t, blockRow, m);
        float* Bsub = GetSubMatrix(b, t, m, blockCol);

        // Load matrices into shared memory
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
        As[row][col] = Asub[row * t + col];
        Bs[row][col] = Bsub[row * t + col];
        __syncthreads();

        // Compute Cvalue using shared memory
        #pragma unroll  // Unroll inner loop for performance
        for (int e = 0; e < BLOCK_SIZE; e++)
        {
            Cvalue += As[row][e] * Bs[e][col];
        }
        __syncthreads();
    }

    Csub[row * t + col] = Cvalue;
}