#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Calculate global thread index
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Check for bounds
    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride + j;
    int image_row_offset = i * image_stride + j;

    // Load flow vectors
    float u_ = u[flow_row_offset];
    float v_ = v[flow_row_offset];

    //Compute target pixel
    float cx = fmaf(u_, time_scale, static_cast<float>(j) + 1.0f);
    float cy = fmaf(v_, time_scale, static_cast<float>(i) + 1.0f);

    // Convert float to integer
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Load source value
    float value = src[image_row_offset];

    // Conditional atomic add
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}