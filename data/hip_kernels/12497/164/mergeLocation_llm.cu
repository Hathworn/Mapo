#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop unrolling for improved performance
    for (int idx = ptidx; idx < npoints; idx += blockDim.x * gridDim.x) 
    {
        short2 loc = loc_[idx];
        x[idx] = loc.x * scale;
        y[idx] = loc.y * scale;
    }
}