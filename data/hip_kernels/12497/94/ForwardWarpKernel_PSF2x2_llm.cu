#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Precompute common values to reduce redundant calculations
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    int tx = (int) px;
    int ty = (int) py;
    float value = src[image_row_offset + j];

    if (tx < w && tx >= 0 && ty < h && ty >= 0) {
        float weight = dx * dy;
        // Combined atomicAdd calls to a single line
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
    
    tx -= 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0) {
        float weight = (1.0f - dx) * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    ty -= 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0) {
        float weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    tx += 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0) {
        float weight = dx * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}