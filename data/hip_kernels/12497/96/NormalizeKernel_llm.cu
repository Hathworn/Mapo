#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate the global thread index in both directions
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to potentially store normalization factors if beneficial (depends on specific use case)
    __shared__ float shared_norm_factor[/* specify size if using shared memory */];

    // Ensure thread indices are within valid range
    if (i < h && j < w) {
        const int pos = i * s + j;

        // Load normalization factor into a variable, can also use shared memory if applicable
        float scale = normalization_factor[pos];

        // Compute inverse scale, handle zero scaling, and apply
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);
        image[pos] *= invScale;
    }
}