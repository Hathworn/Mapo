#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Exit if the thread index is out of bounds
    if (i < h && j < w) {
        int pos = i * w + j;
        image[pos] = value;
    }
}