#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexSelect_contiguous(float *tensor, float *src, long stride, float *index, long idxSize)
{
    const int VT = 4;
    const int WARP_SIZE = 32;
    const int MAX_DIM_SIZE = 65535;

    // Calculate the index based on block and thread info
    int idx = blockIdx.x * blockDim.y + threadIdx.y;
    int startIdx = threadIdx.x + blockIdx.y * VT * WARP_SIZE;

    while (idx < idxSize) {
        while (startIdx < stride) {
            const int srcIdx = ((int)index[idx] - 1) * stride;
            const int targetIdx = idx * stride;

            // Unrolled loop to improve memory coalescing
            #pragma unroll
            for (int i = 0; i < VT; i++) {
                const int featureIdx = startIdx + i * WARP_SIZE;
                if (featureIdx < stride) {
                    tensor[targetIdx + featureIdx] = src[srcIdx + featureIdx];
                }
            }
            startIdx += VT * WARP_SIZE * MAX_DIM_SIZE;
        }
        idx += blockDim.y * MAX_DIM_SIZE;
    }
}