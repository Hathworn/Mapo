#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_renorm(float *data, const float value, const long size, const float maxnorm)
{
    __shared__ float buffer[32];
    long tx = threadIdx.x;
    long bx = blockIdx.x;
    long step = blockDim.x;
    float *row = data + size * bx;

    buffer[tx] = 0.0f;

    // Calculate partial norms in parallel
    for (long i = tx; i < size; i += step)
    {
        buffer[tx] += powf(fabsf(row[i]), value);
    }

    // Parallel reduction to sum the buffer
    __syncthreads();
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        if (tx < stride)
        {
            buffer[tx] += buffer[tx + stride];
        }
        __syncthreads();
    }

    // Clip norms and renormalize if necessary
    if (tx == 0) {
        float norm = powf(buffer[0], 1.0f / value);
        if (norm > maxnorm)
        {
            norm = maxnorm / (norm + 1e-7f);
            buffer[0] = norm;
        }
        else
        {
            buffer[0] = 1.0f;
        }
    }

    __syncthreads();
    float normFactor = buffer[0];

    for (long i = tx; i < size; i += step)
    {
        row[i] *= normFactor;
    }
}