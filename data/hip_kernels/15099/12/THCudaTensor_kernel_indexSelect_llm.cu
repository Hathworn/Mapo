#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexSelect(float *tensor, float *src, long* src_stride, float *index, long src_nDim, int dim, long idx_size, long tensor_size, long size_dim)
{
    // Improve thread index calculation for better mapping across blocks and threads
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

    long flat_size = tensor_size / idx_size;

    if (thread_idx < flat_size)
    {
        long coeff, leftover;
        int targetIdx, srcIdx, stride_d;

        for (int i = 0; i < idx_size; i++)
        {
            leftover = thread_idx;
            targetIdx = 0;
            srcIdx = 0;

            // Use loop unrolling for efficiency and access each dimension
            for (int d = 0; d < dim; d++)
            {
                stride_d = src_stride[d] / size_dim;
                coeff = leftover / stride_d;
                leftover -= coeff * stride_d;
                targetIdx += coeff * stride_d * idx_size;
                srcIdx += coeff * src_stride[d];
            }

            for (int d = dim + 1; d < src_nDim; d++)
            {
                coeff = leftover / src_stride[d];
                leftover -= coeff * src_stride[d];
                targetIdx += coeff * src_stride[d];
                srcIdx += coeff * src_stride[d];
            }

            // Minimize the index calculation to only once in loop iteration
            int index_val = (int)(index[i]) - 1;
            tensor[targetIdx + i * src_stride[dim]] = src[srcIdx + index_val * src_stride[dim]];
        }
    }
}