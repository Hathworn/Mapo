#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_indexFill(float *tensor, long* stride, float *index, long src_nDim, int dim, long idx_size, long tensor_size, long size_dim, float val)
{
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    long flat_size = tensor_size / idx_size;

    // Utilize shared memory for frequently accessed data to reduce global memory accesses
    __shared__ long sharedStride[16];  // Adjust size according to src_nDim if larger
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        for (int d = 0; d < src_nDim; d++) {
            sharedStride[d] = stride[d];
        }
    }
    __syncthreads();

    if (thread_idx < flat_size)
    {
        long coeff = 0;
        for (int i = 0; i < idx_size; i++)
        {
            int leftover = thread_idx;
            int srcIdx = 0;
            for (int d = 0; d < src_nDim; d++)
            {
                // Simplified redundant condition checks
                coeff = leftover / (d == dim ? size_dim : sharedStride[d] / size_dim);
                leftover -= coeff * (d == dim ? size_dim : sharedStride[d] / size_dim);
                srcIdx += coeff * sharedStride[d];
            }
            tensor[srcIdx + (int)((index[i]) - 1) * sharedStride[dim]] = val;
        }
    }
}