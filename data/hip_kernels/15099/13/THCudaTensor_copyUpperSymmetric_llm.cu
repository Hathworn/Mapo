#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_copyUpperSymmetric(float *input, int n, int len)
{
    // Calculate global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Loop across threads by striding with grid size
    while (idx < len) {
        const int r = idx % n;
        const int c = idx / n;
        if (r > c) {
            input[idx] = input[r * n + c];
        }
        // Update index by total number of threads
        idx += blockDim.x * gridDim.x;
    }
}