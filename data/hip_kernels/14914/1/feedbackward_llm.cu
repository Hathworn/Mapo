#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
__device__ __managed__ float weightedinputs[25]; // used as list of input neuron
__device__ __managed__ float weights[25]; // used as list of neuron connection weights
__device__ __managed__ int inputs[25]; // used as list of neuron connection weights
__device__ __managed__ float output = 0; // used to return output
__device__ __managed__ int expctd = 0; // used to return output

__global__ void feedbackward() { // trains the weights
    // Cache frequently accessed variables
    int idx = threadIdx.x; // Cache the thread index
    float lr = 0.3;
    float local_output = output; // Cache output value
    int local_expctd = expctd;   // Cache expected value
    float error = (local_expctd - local_output);

    // Update weights using cached indices
    weights[idx] += error * inputs[idx] * lr;
}