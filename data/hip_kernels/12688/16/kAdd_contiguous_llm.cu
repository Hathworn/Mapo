#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate the thread's unique index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop for better memory access pattern
    for (unsigned int stride = idx; stride < numEls; stride += blockDim.x * gridDim.x) {
        dest[stride] = a[stride] + b[stride];
    }
}