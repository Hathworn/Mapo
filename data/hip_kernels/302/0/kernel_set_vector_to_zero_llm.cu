#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_set_vector_to_zero(double *d_vec, int dimension)
{
    // Calculate global thread ID
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure gid is within bounds and set the vector element to zero
    if (gid < dimension) {
        d_vec[gid] = 0.0;
    }
}