#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void doubleArray2floatArray(const double * doubleArray, float* floatArray, const int size) {
    // Calculate a unique global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Process data in a coalesced manner, handle multiple elements per thread
    for (int idx = i; idx < size; idx += blockDim.x * gridDim.x) {
        floatArray[idx] = __double2float_rd(doubleArray[idx]); // Use intrinsic for conversion
    }
}