#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_mult_vector_by_number(double *vec, double alpha, int numElements)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Combine condition check with loop
    for (int index = gid; index < numElements; index += blockDim.x * gridDim.x) {
        vec[index] *= alpha;
    }
}