#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_vec_equals_minus_vec1(double *vec, double *vec1, int numElements)
{
    // Calculate global thread ID
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform computation only if gid is within bounds
    if (gid < numElements) {
        vec[gid] = -vec1[gid];
    }
}