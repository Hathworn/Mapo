#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void floatArray2doubleArray(const float * floatArray, double* doubleArray, const int size) {
    // Calculate global index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Only convert elements if within bounds
    if (i < size) {
        doubleArray[i] = static_cast<double>(floatArray[i]);
    }
}