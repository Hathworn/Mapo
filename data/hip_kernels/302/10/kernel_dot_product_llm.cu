#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_dot_product(const double * vec1, const double * vec2, int numElements, double * answer)
{
    extern __shared__ double products[]; // Shared memory for partial products

    int i = threadIdx.x;

    // Load and compute product for current element
    if (i < numElements) {
        products[i] = vec1[i] * vec2[i];
    } else {
        products[i] = 0.0; // Handle threads exceeding the number of elements
    }

    __syncthreads();

    // Reduce sum across all threads using sequential addressing
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if (i % (2 * stride) == 0 && (i + stride) < numElements) {
            products[i] += products[i + stride];
        }
        __syncthreads();
    }

    // Write final result to answer
    if (i == 0) {
        *answer = products[0];
    }
}