#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_euclidean_norm(const double *vec, int numElements, double *answer)
{
    extern __shared__ double square[];  // one element per thread

    int i = threadIdx.x;  // numElements assumed to fit into one block
    square[i] = vec[i] * vec[i];

    __syncthreads();

    // Optimize reduction using parallel threads
    for (int stride = 1; stride < numElements; stride *= 2) {
        if (i % (2 * stride) == 0 && i + stride < numElements) {
            square[i] += square[i + stride];
        }
        
        __syncthreads();
    }

    // Store the result to answer
    if (i == 0) {
        *answer = sqrt(square[0]);
    }
}