#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_vec_equals_vec1_plus_alpha_times_vec2(double *vec, double *vec1, double alpha, double *d_a1, double *vec2, int numElements)
{
    // Calculate the global thread ID
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gid < numElements) {
        // Use a single variable for combined alpha calculation
        double effective_alpha = alpha;
        if (d_a1) effective_alpha *= *d_a1;

        // Perform the vector operation
        vec[gid] = vec1[gid] + effective_alpha * vec2[gid];
    }
}