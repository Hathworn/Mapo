#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_add_regularization_term(double* d_input_vector, int dimension, double regularization_parameter, double* d_rv)
{
    // Use shared memory for intermediate sum calculation
    extern __shared__ double shared_sum[];
    int idx = threadIdx.x;

    shared_sum[idx] = 0;
    
    // Calculate partial sum for each thread
    for (int i = idx + 1; i < dimension; i += blockDim.x) {
        shared_sum[idx] += 0.5 * d_input_vector[i] * d_input_vector[i] * regularization_parameter;
    }
    
    __syncthreads();
    
    // Use reduction to sum up partial results
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (idx < stride) {
            shared_sum[idx] += shared_sum[idx + stride];
        }
        __syncthreads();
    }
    
    // One thread writes the result to global memory
    if (idx == 0) {
        atomicAdd(d_rv, shared_sum[0]);
    }
}