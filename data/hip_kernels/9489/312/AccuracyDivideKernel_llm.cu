#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to divide accuracy by N
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use a single thread to perform the division
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}