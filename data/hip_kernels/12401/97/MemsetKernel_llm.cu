#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Use 2D threadblock and grid for better memory access pattern
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < h && j < w) // Combined bounds check for efficiency
    {
        const int pos = i * w + j;
        image[pos] = value;
    }
}