#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate global thread index
    int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Utilize shared memory for efficient data loading
    __shared__ short2 shared_loc[256];
    
    if (ptidx < npoints)
    {
        // Load data into shared memory
        shared_loc[threadIdx.x] = loc_[ptidx];
        __syncthreads(); // Ensure all threads have loaded data
        
        // Perform computation using shared memory
        x[ptidx] = shared_loc[threadIdx.x].x * scale;
        y[ptidx] = shared_loc[threadIdx.x].y * scale;
    }
}