#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    // pixel containing bottom left corner
    float px, py;
    float dx = modff (cx, &px);
    float dy = modff (cy, &py);
    // target pixel integer coords
    int tx = (int) px;
    int ty = (int) py;
    if ((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)) return;

    // Pre-compute repetitive values for efficiency
    float p_dx = 1.0f - dx;
    float p_dy = 1.0f - dy;

    float value = src[image_row_offset + j];
    float weight;

    // fill pixel containing bottom right corner
    weight = dx * dy;
    _atomicAdd(dst + ty * image_stride + tx, value * weight);
    _atomicAdd(normalization_factor + ty * image_stride + tx, weight);

    // fill pixel containing bottom left corner
    tx -= 1;
    if (tx >= 0) // Check bounds once instead of multiple times
    {
        weight = p_dx * dy;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // fill pixel containing upper left corner
    ty -= 1;
    if (ty >= 0 && tx >= 0) // Check bounds once instead of multiple times
    {
        weight = p_dx * p_dy;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // fill pixel containing upper right corner
    tx += 1;
    if (ty >= 0 && tx < w) // Check bounds once instead of multiple times
    {
        weight = dx * p_dy;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}