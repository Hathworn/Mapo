#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved the kernel to eliminate unnecessary execution
__global__ void sync_conv_groups() {
    __syncthreads(); // Ensure all threads within a block are synchronized
}