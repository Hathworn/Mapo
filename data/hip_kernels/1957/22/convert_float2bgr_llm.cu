#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convert_float2bgr(float* annd, unsigned char* bgr, int w, int h, float minval, float maxval)
{
    // Calculate the row and column of the element
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within image bounds
    if (x < w && y < h)
    {
        int id = y * w + x;

        // Streamline calculation, reduce function calls
        float normalizedValue = (annd[id] - minval) / (maxval - minval);
        normalizedValue = fmaxf(fminf(normalizedValue, 1.f), 0.f);

        // Directly assign to bgr with a single operation
        bgr[id] = static_cast<unsigned char>(normalizedValue * 255.f);
    }
}