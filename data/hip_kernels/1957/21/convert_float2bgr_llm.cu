#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convert_float2bgr(float* annd, unsigned char* bgr, int w, int h)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure all threads in block have data stored in shared memory
    __shared__ float shared_annd[1024]; // example size; may need tuning for specific block size

    if (x < w && y < h)
    {
        int id = y * w + x;
        
        // Load to shared memory
        shared_annd[threadIdx.y * blockDim.x + threadIdx.x] = annd[id];
        __syncthreads();

        // Perform computation using shared memory
        int err = max(min((shared_annd[threadIdx.y * blockDim.x + threadIdx.x] + 1.f), 1.f), 0.f) * 255.f;

        // Store computed value
        bgr[id] = err;
    }
}