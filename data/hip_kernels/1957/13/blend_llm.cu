#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blend(float *cmap, float* oldd, float* newd, float weight, int *params)
{
    // Calculate global thread indices
    int ax = blockIdx.x * blockDim.x + threadIdx.x;
    int ay = blockIdx.y * blockDim.y + threadIdx.y;

    // Unpack parameters to local variables
    int ch = params[0];
    int ah = params[1];
    int aw = params[2];

    int slice_a = ah * aw;
    int pitch_a = aw;

    float thre = 0.05; // Threshold value

    if (ax < aw && ay < ah) // Check bounds within the image
    {
        float fa = cmap[ay * pitch_a + ax];

        // Conditionally set blending factor
        fa = (fa < thre) ? 0.0f : weight;

        int baseIndex = ay * pitch_a + ax; // Base index for slice

        // Unroll loop to optimize multiple channel processing
        #pragma unroll
        for (int i = 0; i < ch; i++)
        {
            int idx = i * slice_a + baseIndex;
            newd[idx] = oldd[idx] * fa + newd[idx] * (1.0f - fa);
        }
    }
}