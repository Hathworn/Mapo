#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update2(float *alphaMinusBeta_out, const float *rho, const float *yDotZ, const float *alpha)
{
    // Cache *rho, *yDotZ, *alpha in registers for faster access
    float rho_val = *rho;
    float yDotZ_val = *yDotZ;
    float alpha_val = *alpha;
    
    // Reuse cached values to compute beta
    const float beta = rho_val * yDotZ_val;
    
    // Store the result in the output
    *alphaMinusBeta_out = alpha_val - beta;
}