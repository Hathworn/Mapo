#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update3(float *rho_out, float *H0_out, const float *yDotS, const float *yDotY)
{
    // Load yDotS and yDotY into registers to reduce global memory access
    float yDotS_val = *yDotS;
    float yDotY_val = *yDotY;
    
    // Calculate rho_out
    *rho_out = 1.0f / yDotS_val;
    
    // Conditional calculation of H0_out
    if (yDotY_val > 1e-5)
    {
        *H0_out = yDotS_val / yDotY_val;
    }
}