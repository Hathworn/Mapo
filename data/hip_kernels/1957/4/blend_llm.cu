#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blend(float *cmap, float* oldd, float* newd, float weight, int *params)
{
    // Calculate single thread's global index once
    int ax = blockIdx.x * blockDim.x + threadIdx.x;
    int ay = blockIdx.y * blockDim.y + threadIdx.y;

    // Retrieve parameters to minimize repeated access
    int ch = params[0];
    int ah = params[1];
    int aw = params[2];
    int slice_a = ah * aw;
    int pitch_a = aw;

    // Use constant memory for often used constants
    const float thre = 0.05;

    // Check boundary once at the beginning
    if (ax < aw && ay < ah)
    {
        int idx = ay * pitch_a + ax;           // Compute index outside the loop
        float fa = cmap[idx];
        fa = (fa < thre) ? 0.0f : weight;      // Use conditional operator

        for (int i = 0; i < ch; i++)
        {
            int data_idx = i * slice_a + idx;  // Compute data index once
            float old_val = oldd[data_idx];
            float new_val = newd[data_idx];
            newd[data_idx] = old_val * fa + new_val * (1.0f - fa); // Combine computations
        }
    }
}