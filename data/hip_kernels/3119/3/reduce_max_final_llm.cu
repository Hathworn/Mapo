#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ================================================================
#define THREADS_PER_BLOCK 256
#define MIN_DISTANCE 1.0

__global__ void reduce_max_final(double *g_idata, double *g_odata, int *max_idx, unsigned int n, int blockSize)
{
    __shared__ double sdata[THREADS_PER_BLOCK];
    __shared__ int idxData[THREADS_PER_BLOCK];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + tid;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    double myMax = -1.0;
    int myMaxIndex = -1;

    while (i < n)
    {
        if (myMax < g_idata[i])
        {
            myMax = g_idata[i];
            myMaxIndex = max_idx[i];
        }

        if (i + blockSize < n)
        {
            if (myMax < g_idata[i + blockSize])
            {
                myMax = g_idata[i + blockSize];
                myMaxIndex = max_idx[i + blockSize];
            }
        }

        i += gridSize;
    }

    sdata[tid] = myMax;
    idxData[tid] = myMaxIndex;
    __syncthreads();

    // Utilize loop to reduce code repetition in warp reduction
    for (unsigned int s = blockSize / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            if (sdata[tid] < sdata[tid + s])
            {
                sdata[tid] = sdata[tid + s];
                idxData[tid] = idxData[tid + s];
            }
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
        max_idx[blockIdx.x] = idxData[0];
    }
}