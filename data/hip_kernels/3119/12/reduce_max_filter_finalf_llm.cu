#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256
#define MIN_DISTANCE 1.0

__device__ bool checkDistancef(volatile int *maxesRow, volatile int *maxesCol, int nMax, int curIdx, int width)
{
    int row = curIdx / width;
    int col = curIdx % width;
    for (int j = 0; j < nMax; j++)
    {
        if (maxesRow[j] == row && maxesCol[j] == col)
            return false;
    }
    return true;
}

__global__ void reduce_max_filter_finalf(float *g_idata, float *g_odata, int *max_idx, unsigned int n, unsigned int width, int blockSize, int *maxes, int nMax)
{
    __shared__ int smaxesRow[10];
    __shared__ int smaxesCol[10];
    __shared__ int smaxesVal[10];
    __shared__ float sdata[THREADS_PER_BLOCK];
    __shared__ int idxData[THREADS_PER_BLOCK];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + tid;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    if (tid < nMax)
    {
        smaxesVal[tid] = maxes[tid];
        smaxesRow[tid] = smaxesVal[tid] / width;
        smaxesCol[tid] = smaxesVal[tid] % width;
    }
    __syncthreads();

    float myMax = 0.0;
    int myMaxIndex;

    while (i < n)
    {
        if (myMax < g_idata[i] && checkDistancef(smaxesRow, smaxesCol, nMax, max_idx[i], width))
        {
            myMax = g_idata[i];
            myMaxIndex = max_idx[i];
        }

        if (i + blockSize < n && myMax < g_idata[i + blockSize] && checkDistancef(smaxesRow, smaxesCol, nMax, max_idx[i + blockSize], width))
        {
            myMax = g_idata[i + blockSize];
            myMaxIndex = max_idx[i + blockSize];
        }

        i += gridSize; 
    }

    sdata[tid] = myMax;
    idxData[tid] = myMaxIndex;

    __syncthreads();

    // Start reduction
    if (blockSize >= 512 && tid < 256)
    {
        if (sdata[tid] < sdata[tid + 256] && checkDistancef(smaxesRow, smaxesCol, nMax, idxData[tid + 256], width))
        {
            sdata[tid] = sdata[tid + 256];
            idxData[tid] = idxData[tid + 256];
        }
        __syncthreads();
    }

    if (blockSize >= 256 && tid < 128)
    {
        if (sdata[tid] < sdata[tid + 128] && checkDistancef(smaxesRow, smaxesCol, nMax, idxData[tid + 128], width))
        {
            sdata[tid] = sdata[tid + 128];
            idxData[tid] = idxData[tid + 128];
        }
        __syncthreads();
    }

    if (blockSize >= 128 && tid < 64)
    {
        if (sdata[tid] < sdata[tid + 64] && checkDistancef(smaxesRow, smaxesCol, nMax, idxData[tid + 64], width))
        {
            sdata[tid] = sdata[tid + 64];
            idxData[tid] = idxData[tid + 64];
        }
        __syncthreads();
    }

    volatile float *vdata = sdata;
    volatile int *vidxData = idxData;
    volatile int *vsmaxesRow = smaxesRow;
    volatile int *vsmaxesCol = smaxesCol;

    if (tid < 32)
    {
        // Unroll last warp using volatile memory
        if (blockSize >= 64)
        {
            if (vdata[tid] < vdata[tid + 32] && checkDistancef(vsmaxesRow, vsmaxesCol, nMax, vidxData[tid + 32], width))
            {
                vdata[tid] = vdata[tid + 32];
                vidxData[tid] = vidxData[tid + 32];
            }
        }
        if (blockSize >= 32)
        {
            if (vdata[tid] < vdata[tid + 16] && checkDistancef(vsmaxesRow, vsmaxesCol, nMax, vidxData[tid + 16], width))
            {
                vdata[tid] = vdata[tid + 16];
                vidxData[tid] = vidxData[tid + 16];
            }
        }
        if (blockSize >= 16)
        {
            if (vdata[tid] < vdata[tid + 8] && checkDistancef(vsmaxesRow, vsmaxesCol, nMax, vidxData[tid + 8], width))
            {
                vdata[tid] = vdata[tid + 8];
                vidxData[tid] = vidxData[tid + 8];
            }
        }
        if (blockSize >= 8)
        {
            if (vdata[tid] < vdata[tid + 4] && checkDistancef(vsmaxesRow, vsmaxesCol, nMax, vidxData[tid + 4], width))
            {
                vdata[tid] = vdata[tid + 4];
                vidxData[tid] = vidxData[tid + 4];
            }
        }
        if (blockSize >= 4)
        {
            if (vdata[tid] < vdata[tid + 2] && checkDistancef(vsmaxesRow, vsmaxesCol, nMax, vidxData[tid + 2], width))
            {
                vdata[tid] = vdata[tid + 2];
                vidxData[tid] = vidxData[tid + 2];
            }
        }
        if (blockSize >= 2)
        {
            if (vdata[tid] < vdata[tid + 1] && checkDistancef(vsmaxesRow, vsmaxesCol, nMax, vidxData[tid + 1], width))
            {
                vdata[tid] = vdata[tid + 1];
                vidxData[tid] = vidxData[tid + 1];
            }
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
        max_idx[blockIdx.x] = idxData[0];

        if (gridDim.x == 1)
            maxes[nMax] = idxData[0];
    }
}