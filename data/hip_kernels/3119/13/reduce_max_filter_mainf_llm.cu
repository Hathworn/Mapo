#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256
#define MIN_DISTANCE 1.0

// Efficient warp reduction using shared memory optimizations.
__global__ void reduce_max_filter_mainf(float *g_idata, float *g_odata, int * max_idx, unsigned int width, unsigned int height, int blockSize, int *maxes, int nMax)
{
    __shared__ int smaxesRow[10];
    __shared__ int smaxesCol[10];
    __shared__ int smaxesVal[10];
    __shared__ float sdata[THREADS_PER_BLOCK];
    __shared__ int idxData[THREADS_PER_BLOCK];
    
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize) + tid;
    unsigned int gridSize = blockSize * gridDim.x;

    if (tid < nMax)
    {
        smaxesVal[tid] = maxes[tid];
        smaxesRow[tid] = smaxesVal[tid] / width;
        smaxesCol[tid] = smaxesVal[tid] % width;
    }
    __syncthreads();

    float myMax = -INFINITY;
    int myMaxIndex;
    float val;

    while (i < width * height)
    {
        val = g_idata[i];
        if (myMax < val)
        {
            if (checkDistancef(smaxesRow, smaxesCol, nMax, i, width))
            {
                myMax = val;
                myMaxIndex = i;
            }
        }

        if (i + blockSize < width * height)
        {
            val = g_idata[i + blockSize];
            if (myMax < val)
            {
                if (checkDistancef(smaxesRow, smaxesCol, nMax, i + blockSize, width))
                {
                    myMax = val;
                    myMaxIndex = i + blockSize;
                }
            }
        }

        i += gridSize;
    }

    sdata[tid] = myMax;
    idxData[tid] = myMaxIndex;
    __syncthreads();

    // Improved reduction using a single warp
    if (tid < 32)
    {
        volatile float *vdata = sdata;
        volatile int *vidxData = idxData;

        for (int offset = 32; offset >= 1; offset /= 2)
        {
            if (myMax < vdata[tid + offset])
            {
                if (checkDistancef(smaxesRow, smaxesCol, nMax, vidxData[tid + offset], width))
                {
                    vdata[tid] = myMax = vdata[tid + offset];
                    vidxData[tid] = vidxData[tid + offset];
                }
            }
        }
    }

    // Only the first thread in a block writes the result
    if (tid == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
        max_idx[blockIdx.x] = idxData[0];

        if (gridDim.x == 1)
            maxes[nMax] = idxData[0];
    }
}
```
