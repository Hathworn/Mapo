#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256
#define MIN_DISTANCE 1.0

__global__ void reduce_max_mainf(float *g_idata, float *g_odata, int *max_idx, unsigned int n, int blockSize)
{
    __shared__ float sdata[THREADS_PER_BLOCK];
    __shared__ int idxData[THREADS_PER_BLOCK];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockSize + tid;
    unsigned int gridSize = blockSize * gridDim.x;

    float myMax = -FLT_MAX; // Initialize to a very small number
    int myMaxIndex = -1;

    // Unrolling the loop and handling boundaries
    while (i < n) {
        float val = g_idata[i];
        if (myMax < val) {
            myMax = val;
            myMaxIndex = i;
        }
        i += gridSize;
    }

    sdata[tid] = myMax;
    idxData[tid] = myMaxIndex;
    __syncthreads();

    // Reduce using a single loop
    for (unsigned int s = blockSize / 2; s > 32; s >>= 1) {
        if (tid < s) {
            if (sdata[tid] < sdata[tid + s]) {
                sdata[tid] = sdata[tid + s];
                idxData[tid] = idxData[tid + s];
            }
        }
        __syncthreads();
    }

    // Warp reduction without __syncthreads
    if (tid < 32) {
        volatile float *vdata = sdata;
        volatile int *vidxData = idxData;
        if (vdata[tid] < vdata[tid + 32]) {
            vdata[tid] = vdata[tid + 32];
            vidxData[tid] = vidxData[tid + 32];
        }
        if (vdata[tid] < vdata[tid + 16]) {
            vdata[tid] = vdata[tid + 16];
            vidxData[tid] = vidxData[tid + 16];
        }
        if (vdata[tid] < vdata[tid + 8]) {
            vdata[tid] = vdata[tid + 8];
            vidxData[tid] = vidxData[tid + 8];
        }
        if (vdata[tid] < vdata[tid + 4]) {
            vdata[tid] = vdata[tid + 4];
            vidxData[tid] = vidxData[tid + 4];
        }
        if (vdata[tid] < vdata[tid + 2]) {
            vdata[tid] = vdata[tid + 2];
            vidxData[tid] = vidxData[tid + 2];
        }
        if (vdata[tid] < vdata[tid + 1]) {
            vdata[tid] = vdata[tid + 1];
            vidxData[tid] = vidxData[tid + 1];
        }
    }

    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
        max_idx[blockIdx.x] = idxData[0];
    }
}