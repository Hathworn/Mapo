#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256
#define MIN_DISTANCE 1.0

// ================================================================
__global__ void reduce_max_finalf(float *g_idata, float *g_odata, int * max_idx, unsigned int n, int blockSize)
{
    __shared__ float sdata[THREADS_PER_BLOCK];
    __shared__ int idxData[THREADS_PER_BLOCK];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + tid;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    float myMax = 0.0;
    int myMaxIndex = -1;  // Initializing index for safety

    // Loop over elements in strides
    while (i < n)
    {
        if (myMax < g_idata[i])
        {
            myMax = g_idata[i];
            myMaxIndex = max_idx[i];
        }
        if (i + blockSize < n)
        {
            if (myMax < g_idata[i + blockSize])
            {
                myMax = g_idata[i + blockSize];
                myMaxIndex = max_idx[i + blockSize];
            }
        }
        i += gridSize;
    }

    sdata[tid] = myMax;
    idxData[tid] = myMaxIndex;

    __syncthreads();

    // Use a loop for reductions instead of unrolling
    for (unsigned int stride = blockSize / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            if (sdata[tid] < sdata[tid + stride])
            {
                sdata[tid] = sdata[tid + stride];
                idxData[tid] = idxData[tid + stride];
            }
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
        max_idx[blockIdx.x] = idxData[0];
    }
}