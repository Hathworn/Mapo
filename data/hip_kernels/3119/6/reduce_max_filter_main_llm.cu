#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256
#define MIN_DISTANCE 1.0

__device__ bool checkDistance(int *maxesRow, int *maxesCol, int nMax, int curIdx, int width)
{
    int row = curIdx / width;
    int col = curIdx % width;
    for (int j = 0; j < nMax; j++)
    {
        if (maxesRow[j] == row && maxesCol[j] == col)
            return false;
    }
    return true;
}

__device__ double distance(int x1, int x2, int y1, int y2) 
{
    return ((double(x1-x2))*(double(x1-x2))) + 
           ((double(y1-y2))*(double(y1-y2)));
}

__global__ void reduce_max_filter_main(double *g_idata, double *g_odata, int *max_idx, unsigned int width, unsigned int height, int blockSize, int *maxes, int nMax)
{
    __shared__ int smaxesRow[10];
    __shared__ int smaxesCol[10];
    __shared__ int smaxesVal[10];
    __shared__ double sdata[THREADS_PER_BLOCK];
    __shared__ int idxData[THREADS_PER_BLOCK];
    
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize) + tid;
    unsigned int gridSize = blockSize * gridDim.x;

    // Initialize shared memory
    if (tid < nMax)
    {
        smaxesVal[tid] = maxes[tid];
        smaxesRow[tid] = smaxesVal[tid] / width;
        smaxesCol[tid] = smaxesVal[tid] % width;
    }
    __syncthreads(); // Ensure all threads initialize shared memory

    double myMax = -INFINITY;
    int myMaxIndex;
    double val;

    // Unrolled loop for better performance
    while (i < width * height)
    {
        val = g_idata[i];
        if (myMax < val)
        {
            if (checkDistance(smaxesRow, smaxesCol, nMax, i, width))
            {
                myMax = val;
                myMaxIndex = i;
            }
        }
        
        int nextIdx = i + blockSize;
        if (nextIdx < width * height)
        {
            val = g_idata[nextIdx];
            if (myMax < val)
            {
                if (checkDistance(smaxesRow, smaxesCol, nMax, nextIdx, width))
                {
                    myMax = val;
                    myMaxIndex = nextIdx;
                }
            }
        }
        
        i += gridSize;
    }

    sdata[tid] = myMax;
    idxData[tid] = myMaxIndex;

    __syncthreads(); // Synchronize before reduction

    // Begin reduction with loop unrolling
    for (int s = blockSize / 2; s > 32; s >>= 1)
    {
        if (tid < s)
        {
            if (sdata[tid] < sdata[tid + s])
            {
                if (checkDistance(smaxesRow, smaxesCol, nMax, idxData[tid + s], width))
                {
                    sdata[tid] = sdata[tid + s];
                    idxData[tid] = idxData[tid + s];
                }
            }
        }
        __syncthreads();
    }

    // Warp-level logic for the last 32 threads
    if (tid < 32)
    {
        volatile double *vdata = sdata;
        volatile int *vidxData = idxData;
        
        if (vdata[tid] < vdata[tid + 32])
        {
            if (checkDistance(smaxesRow, smaxesCol, nMax, vidxData[tid + 32], width))
            {
                vdata[tid] = vdata[tid + 32];
                vidxData[tid] = vidxData[tid + 32];
            }
        }
        if (vdata[tid] < vdata[tid + 16])
        {
            if (checkDistance(smaxesRow, smaxesCol, nMax, vidxData[tid + 16], width))
            {
                vdata[tid] = vdata[tid + 16];
                vidxData[tid] = vidxData[tid + 16];
            }
        }
        if (vdata[tid] < vdata[tid + 8])
        {
            if (checkDistance(smaxesRow, smaxesCol, nMax, vidxData[tid + 8], width))
            {
                vdata[tid] = vdata[tid + 8];
                vidxData[tid] = vidxData[tid + 8];
            }
        }
        if (vdata[tid] < vdata[tid + 4])
        {
            if (checkDistance(smaxesRow, smaxesCol, nMax, vidxData[tid + 4], width))
            {
                vdata[tid] = vdata[tid + 4];
                vidxData[tid] = vidxData[tid + 4];
            }
        }
        if (vdata[tid] < vdata[tid + 2])
        {
            if (checkDistance(smaxesRow, smaxesCol, nMax, vidxData[tid + 2], width))
            {
                vdata[tid] = vdata[tid + 2];
                vidxData[tid] = vidxData[tid + 2];
            }
        }
        if (vdata[tid] < vdata[tid + 1])
        {
            if (checkDistance(smaxesRow, smaxesCol, nMax, vidxData[tid + 1], width))
            {
                vdata[tid] = vdata[tid + 1];
                vidxData[tid] = vidxData[tid + 1];
            }
        }
    }
    
    // Write result for this block to global memory
    if (tid == 0)
    {
        g_odata[blockIdx.x] = sdata[0];
        max_idx[blockIdx.x] = idxData[0];
        
        if (gridDim.x == 1)
            maxes[nMax] = idxData[0];
    }
}