#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256
#define MIN_DISTANCE 1.0

__device__ bool checkDistance(volatile int *maxesRow, volatile int *maxesCol, int nMax, int curIdx, int width) {
    int row = curIdx / width;
    int col = curIdx % width;
    for (int j = 0; j < nMax; j++) {
        if (maxesRow[j] == row && maxesCol[j] == col)
            return false;
    }
    return true;
}

__global__ void reduce_max_filter_final(double *g_idata, double *g_odata, int *max_idx, unsigned int n, unsigned int width, int blockSize, int *maxes, int nMax) {
    __shared__ int smaxesRow[10];
    __shared__ int smaxesCol[10];
    __shared__ double sdata[THREADS_PER_BLOCK];
    __shared__ int idxData[THREADS_PER_BLOCK];
    
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + tid;
    unsigned int gridSize = blockSize * 2 * gridDim.x;

    // Load shared memory with max index information
    if (tid < nMax) {
        smaxesRow[tid] = maxes[tid] / width;
        smaxesCol[tid] = maxes[tid] % width;
    }
    __syncthreads();

    double myMax = 0.0;
    int myMaxIndex = -1;  // Initialize index to invalid value

    // Unrolling loop by two
    while (i < n) {
        if (myMax < g_idata[i] && checkDistance(smaxesRow, smaxesCol, nMax, max_idx[i], width)) {
            myMax = g_idata[i];
            myMaxIndex = max_idx[i];
        }
        if (i + blockSize < n && myMax < g_idata[i + blockSize] && checkDistance(smaxesRow, smaxesCol, nMax, max_idx[i + blockSize], width)) {
            myMax = g_idata[i + blockSize];
            myMaxIndex = max_idx[i + blockSize];
        }
        i += gridSize;
    }

    sdata[tid] = myMax;
    idxData[tid] = myMaxIndex;
    __syncthreads();

    // Reduce in shared memory
    if (blockSize >= 512) {
        if (tid < 256 && sdata[tid] < sdata[tid + 256] && checkDistance(smaxesRow, smaxesCol, nMax, idxData[tid + 256], width)) {
            sdata[tid] = sdata[tid + 256];
            idxData[tid] = idxData[tid + 256];
        }
        __syncthreads();
    }
    
    if (blockSize >= 256) {
        if (tid < 128 && sdata[tid] < sdata[tid + 128] && checkDistance(smaxesRow, smaxesCol, nMax, idxData[tid + 128], width)) {
            sdata[tid] = sdata[tid + 128];
            idxData[tid] = idxData[tid + 128];
        }
        __syncthreads();
    }
    
    if (blockSize >= 128) {
        if (tid < 64 && sdata[tid] < sdata[tid + 64] && checkDistance(smaxesRow, smaxesCol, nMax, idxData[tid + 64], width)) {
            sdata[tid] = sdata[tid + 64];
            idxData[tid] = idxData[tid + 64];
        }
        __syncthreads();
    }

    if (tid < 32) {
        volatile double *vdata = sdata;
        volatile int *vidxData = idxData;
        if (blockSize >= 64 && vdata[tid] < vdata[tid + 32] && checkDistance(smaxesRow, smaxesCol, nMax, vidxData[tid + 32], width)) {
            vdata[tid] = vdata[tid + 32];
            vidxData[tid] = vidxData[tid + 32];
        }
        if (blockSize >= 32 && vdata[tid] < vdata[tid + 16] && checkDistance(smaxesRow, smaxesCol, nMax, vidxData[tid + 16], width)) {
            vdata[tid] = vdata[tid + 16];
            vidxData[tid] = vidxData[tid + 16];
        }
        if (blockSize >= 16 && vdata[tid] < vdata[tid + 8] && checkDistance(smaxesRow, smaxesCol, nMax, vidxData[tid + 8], width)) {
            vdata[tid] = vdata[tid + 8];
            vidxData[tid] = vidxData[tid + 8];
        }
        if (blockSize >= 8 && vdata[tid] < vdata[tid + 4] && checkDistance(smaxesRow, smaxesCol, nMax, vidxData[tid + 4], width)) {
            vdata[tid] = vdata[tid + 4];
            vidxData[tid] = vidxData[tid + 4];
        }
        if (blockSize >= 4 && vdata[tid] < vdata[tid + 2] && checkDistance(smaxesRow, smaxesCol, nMax, vidxData[tid + 2], width)) {
            vdata[tid] = vdata[tid + 2];
            vidxData[tid] = vidxData[tid + 2];
        }
        if (blockSize >= 2 && vdata[tid] < vdata[tid + 1] && checkDistance(smaxesRow, smaxesCol, nMax, vidxData[tid + 1], width)) {
            vdata[tid] = vdata[tid + 1];
            vidxData[tid] = vidxData[tid + 1];
        }
    }

    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
        max_idx[blockIdx.x] = idxData[0];

        if (gridDim.x == 1)
            maxes[nMax] = idxData[0];
    }
}