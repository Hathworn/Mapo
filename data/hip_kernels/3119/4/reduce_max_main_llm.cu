#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ================================================================

// Author: Timothy Blattner
// Date:   Wed Nov 30 12:36:40 2011 EScufftDoubleComplex

// ================================================================

#define THREADS_PER_BLOCK 256
#define MIN_DISTANCE 1.0

// ================================================================
__global__ void reduce_max_main(double *g_idata, double *g_odata, int * max_idx, unsigned int n, int blockSize)
{
    __shared__ double sdata[THREADS_PER_BLOCK];
    __shared__ int idxData[THREADS_PER_BLOCK];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockSize + tid;
    unsigned int gridSize = blockSize * gridDim.x;

    double myMax = 0.0;
    int myMaxIndex = -1; // Initialize with invalid index
    double val;

    // Process all elements in the block
    while (i < n)
    {
        val = g_idata[i];
        if (myMax < val)
        {
            myMax = val;
            myMaxIndex = i;
        }
        i += gridSize;
    }

    // Store local max and index to shared memory
    sdata[tid] = myMax;
    idxData[tid] = myMaxIndex;

    __syncthreads();

    // Reduce using shared memory
    if (blockSize >= 512) {
        if (tid < 256) {
            if (sdata[tid] < sdata[tid + 256]) {
                sdata[tid] = sdata[tid + 256];
                idxData[tid] = idxData[tid + 256];
            }
        }
        __syncthreads();
    }

    if (blockSize >= 256) {
        if (tid < 128) {
            if (sdata[tid] < sdata[tid + 128]) {
                sdata[tid] = sdata[tid + 128];
                idxData[tid] = idxData[tid + 128];
            }
        }
        __syncthreads();
    }

    if (blockSize >= 128) {
        if (tid < 64) {
            if (sdata[tid] < sdata[tid + 64]) {
                sdata[tid] = sdata[tid + 64];
                idxData[tid] = idxData[tid + 64];
            }
        }
        __syncthreads();
    }

    // Unrolling last warp for better performance
    if (tid < 32) {
        volatile double *vdata = sdata;
        volatile int *vidxData = idxData;
        if (sdata[tid] < vdata[tid + 32]) {
            vdata[tid] = vdata[tid + 32];
            vidxData[tid] = vidxData[tid + 32];
        }
        if (sdata[tid] < vdata[tid + 16]) {
            vdata[tid] = vdata[tid + 16];
            vidxData[tid] = vidxData[tid + 16];
        }
        if (sdata[tid] < vdata[tid + 8]) {
            vdata[tid] = vdata[tid + 8];
            vidxData[tid] = vidxData[tid + 8];
        }
        if (sdata[tid] < vdata[tid + 4]) {
            vdata[tid] = vdata[tid + 4];
            vidxData[tid] = vidxData[tid + 4];
        }
        if (sdata[tid] < vdata[tid + 2]) {
            vdata[tid] = vdata[tid + 2];
            vidxData[tid] = vidxData[tid + 2];
        }
        if (sdata[tid] < vdata[tid + 1]) {
            vdata[tid] = vdata[tid + 1];
            vidxData[tid] = vidxData[tid + 1];
        }
    }

    // Writing result back to the global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
        max_idx[blockIdx.x] = idxData[0];
    }
}