#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int width = 567;
const int height = 985;

__global__ void testKernel2D(float* M, float* N, float* P, size_t pitch) {
    // Calculate col and row using threadIdx and blockIdx
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensures thread operates within bounds
    if (row < height && col < width) {
        float* row_M = (float*)((char*)M + row * pitch);
        float* row_N = (float*)((char*)N + row * pitch);
        float* row_P = (float*)((char*)P + row * pitch);

        // Perform element-wise addition
        row_P[col] = row_M[col] + row_N[col];
    }
}