#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculateGaussianKernel(float *gaussKernel, const float sigma, int halfKernelWidth){

    int i = threadIdx.x - halfKernelWidth;
    extern __shared__ float s_gaussKernel[];
    
    // Precompute constant values to avoid repeated calculations
    float constantFactor = __fdividef(1, (sqrtf(2 * M_PI * sigma)));
    float variance = 2 * sigma * sigma;

    // Compute Gaussian value
    s_gaussKernel[threadIdx.x] = constantFactor * expf(-1 * __fdividef((i * i), variance));

    __syncthreads();

    // Use reduction pattern for summing the kernel values
    float sum = 0.0f;
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int index = 2 * stride * threadIdx.x;
        if (index < blockDim.x) {
            sum += s_gaussKernel[index + stride];
        }
        __syncthreads();
    }

    // Only one thread should write the normalized kernel value
    if (threadIdx.x == 0) {
        gaussKernel[threadIdx.x] = s_gaussKernel[threadIdx.x] / sum;
    }
}