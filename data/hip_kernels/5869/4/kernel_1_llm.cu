#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_1(float *d_data_in, float *d_data_out, int data_size)
{
    __shared__ float s_data[BLKSIZE];
    int tid = threadIdx.x;
    int index = tid + blockIdx.x * blockDim.x;
    
    // Initialize shared memory only if within bounds
    s_data[tid] = (index < data_size) ? d_data_in[index] : 0.0f;
    __syncthreads();

    // Optimized parallel reduction
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads();
    }

    // Write the result from the first thread
    if (tid == 0) {
        d_data_out[blockIdx.x] = s_data[0];
    }
}