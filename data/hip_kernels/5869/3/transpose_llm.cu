#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose(double *in_d, double *out_d, int row, int col)
{
    // Using shared memory for coalesced access
    __shared__ double tile[32][32];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < row && y < col) {
        tile[threadIdx.y][threadIdx.x] = in_d[x + row * y];
    }

    __syncthreads();

    x = blockIdx.y * blockDim.y + threadIdx.x; // Transpose indices
    y = blockIdx.x * blockDim.x + threadIdx.y;

    if (x < col && y < row) {
        out_d[y + col * x] = tile[threadIdx.x][threadIdx.y];
    }
}