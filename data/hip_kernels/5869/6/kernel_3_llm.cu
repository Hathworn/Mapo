#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_3(float *d_data_in, float *d_data_out, int data_size)
{
  __shared__ float s_data[BLKSIZE];
  int tid = threadIdx.x;
  int index = tid + blockIdx.x * blockDim.x;

  // Initialize shared memory only if within data_size
  if (index < data_size) {
    s_data[tid] = d_data_in[index];
  } else {
    s_data[tid] = 0.0;
  }
  __syncthreads();

  // Use efficient reduction within shared memory
  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      s_data[tid] += s_data[tid + s];
    }
    __syncthreads();
  }

  // Write the result from the first thread in the block
  if (tid == 0) {
    d_data_out[blockIdx.x] = s_data[0];
  }
}