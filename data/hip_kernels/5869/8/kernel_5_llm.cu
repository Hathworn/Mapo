#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_5(float *d_data_in, float *d_data_out, int data_size)
{
    __shared__ float s_data[BLKSIZE]; // Use shared memory without 'volatile'
    int tid = threadIdx.x;
    int index = tid + blockIdx.x * blockDim.x * 2;

    float temp_sum = 0.0f; // Use a local variable to store temporary sum

    if (index < data_size){
        temp_sum = d_data_in[index];
    }
    if ((index + blockDim.x) < data_size){
        temp_sum += d_data_in[index + blockDim.x];
    }
    s_data[tid] = temp_sum; // Store it once to shared memory
    __syncthreads();

    for (int s = blockDim.x / 2; s >= 64; s >>= 1){
        if (tid < s) {
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads();
    }

    // Use warp shuffle operations to optimize reduction within a warp
    if (tid < 32){
        for (int s = 32; s > 0; s >>= 1) {
            s_data[tid] += __shfl_down(s_data[tid], s, 32);
        }
    }

    if (tid == 0){
        d_data_out[blockIdx.x] = s_data[tid];
    }
}