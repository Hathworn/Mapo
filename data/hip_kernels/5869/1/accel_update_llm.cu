#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define min(X,Y) ((X) < (Y) ? (X) : (Y))

__global__ void accel_update(int nx, int ny, double dx2inv, double dy2inv, double* d_z, double* d_a) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Calculate row and column indices
    int r = i / nx;
    int c = i % nx;

    if (i < nx * ny) {
        // Check bounds within the grid
        if (r > 0 && r < ny - 1 && c > 0 && c < nx - 1) {
            // Precompute reused array accesses
            double current_z = d_z[i];
            double ax = (d_z[i + nx] + d_z[i - nx] - 2.0 * current_z) * dx2inv;
            double ay = (d_z[i + 1] + d_z[i - 1] - 2.0 * current_z) * dy2inv;
            d_a[i] = 0.5 * (ax + ay); // Optimize division by 2
        } else {
            d_a[i] = 0.0;
        }
    }
}