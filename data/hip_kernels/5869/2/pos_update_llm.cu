#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define min(X,Y) ((X) < (Y) ? (X) : (Y))

__global__ void pos_update(int nx, int ny, double dt, double* d_z, double* d_v, double* d_a) {
    // Calculate the global index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Bound check for valid threads within matrix boundaries
    if (i < nx * ny) {
        int r = i / nx;
        int c = i % nx;
        
        // Perform computations only if within valid range
        if (r < ny - 1 && r > 0 && c < nx - 1 && c > 0) {
            double v = d_v[i];   // Cache data in registers
            double a = d_a[i];   // Cache data in registers
            v = v + dt * a;
            d_v[i] = v;
            d_z[i] = d_z[i] + dt * v;
        }
    }
}