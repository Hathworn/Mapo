#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_4(float *d_data_in, float *d_data_out, int data_size)
{
    extern __shared__ float s_data[]; // Use dynamic shared memory
    int tid = threadIdx.x;
    int index = tid + blockIdx.x * blockDim.x;

    s_data[tid] = (index < data_size) ? d_data_in[index] : 0.0f; // Direct initialization
    __syncthreads();

    for (int s = blockDim.x / 2; s > 32; s >>= 1) { // Optimize loop for power of two
        if (tid < s) {
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads();
    }

    // Unroll the last warp
    if (tid < 32) {
        volatile float *v_s_data = s_data; // Employ volatile for warp-level operations
        v_s_data[tid] += v_s_data[tid + 32];
        v_s_data[tid] += v_s_data[tid + 16];
        v_s_data[tid] += v_s_data[tid + 8];
        v_s_data[tid] += v_s_data[tid + 4];
        v_s_data[tid] += v_s_data[tid + 2];
        v_s_data[tid] += v_s_data[tid + 1];
    }

    if (tid == 0) {
        d_data_out[blockIdx.x] = s_data[0]; // Store the result
    }
}