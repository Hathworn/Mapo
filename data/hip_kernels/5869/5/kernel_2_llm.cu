#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_2(float *d_data_in, float *d_data_out, int data_size)
{
    __shared__ float s_data[BLKSIZE];
    int tid = threadIdx.x;
    int index = tid + blockIdx.x * blockDim.x;
    
    // Load input data into shared memory
    s_data[tid] = (index < data_size) ? d_data_in[index] : 0.0;
    __syncthreads();

    // Perform reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads();
    }

    // Write the result for this block to output
    if (tid == 0) {
        d_data_out[blockIdx.x] = s_data[0];
    }
}