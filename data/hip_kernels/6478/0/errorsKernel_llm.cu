#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

const int sizePoint = 500;
const int sizeIndividum = 1000;
const int mathValueMutation = 5;
const float dispersionMutation = 5.0f;
const int powCount = 3;
const float randMaxCount = 20.0f;
const int maxPokoleney = 30;

__global__ void errorsKernel(float *points, float *individs, float *errors, int powCount, int sizePoint)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index
    if (id >= sizeIndividum) return; // Ensure within bounds

    float error = 0.0f; // Initialize error for this individual
    for (int i = 0; i < sizePoint; i++) {
        float ans = 0.0f; 
        float x = 1.0f;
        for (int j = 0; j < powCount; j++) {
            if (j > 0) { // Use efficient power calculation
                x *= (float)i;
            }
            ans += x * individs[id * powCount + j];
        }
        float diff = points[i] - ans;
        error += diff * diff;
    }
    errors[id] = sqrt(error); // Calculate the final error
}
```
