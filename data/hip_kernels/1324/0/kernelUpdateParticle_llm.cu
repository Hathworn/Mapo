#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelUpdateParticle(float *positions, float *velocities, float *pBests, float *gBest, float r1, float r2)
{
    // Use shared memory for gBest to reduce global memory access
    __shared__ float sharedGBest[NUM_OF_DIMENSIONS];
    int dim = threadIdx.x % NUM_OF_DIMENSIONS;

    // Load global memory to shared memory
    if (threadIdx.x < NUM_OF_DIMENSIONS) {
        sharedGBest[threadIdx.x] = gBest[threadIdx.x];
    }
    __syncthreads(); // Ensure all threads have updated shared memory

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= NUM_OF_PARTICLES * NUM_OF_DIMENSIONS)
        return;

    float rp = r1;
    float rg = r2;

    // Use shared memory for gBest access
    velocities[i] = OMEGA * velocities[i] + c1 * rp * (pBests[i] - positions[i]) + c2 * rg * (sharedGBest[dim] - positions[i]);
    positions[i] += velocities[i];
}