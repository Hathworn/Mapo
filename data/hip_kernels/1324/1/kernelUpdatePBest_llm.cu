#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float fitness_function(float x[])
{
    float y, yp;
    float res = 0;
    float y1 = 1 + (x[0] - 1) / 4;
    float yn = 1 + (x[NUM_OF_DIMENSIONS - 1] - 1) / 4;

    res += pow(sin(phi * y1), 2) + pow(yn - 1, 2);

    for (int i = 0; i < NUM_OF_DIMENSIONS - 1; i++)
    {
        y = 1 + (x[i] - 1) / 4;
        yp = 1 + (x[i + 1] - 1) / 4;
        res += pow(y - 1, 2) * (1 + 10 * pow(sin(phi * yp), 2));
    }

    return res;
}

__global__ void kernelUpdatePBest(float *positions, float *pBests, float *gBest)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= NUM_OF_PARTICLES * NUM_OF_DIMENSIONS || i % NUM_OF_DIMENSIONS != 0)
        return;

    float tempParticle1[NUM_OF_DIMENSIONS];
    float tempParticle2[NUM_OF_DIMENSIONS];

    // Use shared memory to store fitness values
    __shared__ float sharedFitnessVals[2]; 
    sharedFitnessVals[0] = 0.0f;
    sharedFitnessVals[1] = 0.0f;
    __syncthreads();
    
    for (int j = 0; j < NUM_OF_DIMENSIONS; j++)
    {
        tempParticle1[j] = positions[i + j];
        tempParticle2[j] = pBests[i + j];
    }

    sharedFitnessVals[0] = fitness_function(tempParticle1);
    sharedFitnessVals[1] = fitness_function(tempParticle2);
    __syncthreads();

    if (sharedFitnessVals[0] < sharedFitnessVals[1])
    {
        for (int j = 0; j < NUM_OF_DIMENSIONS; j++)
        {
            pBests[i + j] = tempParticle1[j];
        }

        // Utilize shared memory to store fitness of global best
        sharedFitnessVals[1] = fitness_function(gBest);
        __syncthreads();

        if (sharedFitnessVals[0] < sharedFitnessVals[1])
        {
            for (int j = 0; j < NUM_OF_DIMENSIONS; j++)
            {
                // Atomic exchange for gBest to ensure correctness in concurrent writes
                atomicExch(gBest + j, tempParticle1[j]);
            }
        }
    }
}