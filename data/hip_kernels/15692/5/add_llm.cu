#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Calculate unique thread index
    int threadID = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure kernel handles all elements by iterating in steps
    for (int idx = threadID; idx < N; idx += blockDim.x * gridDim.x)
    {
        c[idx] = a[idx] + b[idx]; // Perform addition
    }
}