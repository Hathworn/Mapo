#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(unsigned char *ptr, int ticks)
{
    // Calculate global x and y indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Compute offset once to avoid redundant calculations
    int offset = x + y * blockDim.x * gridDim.x;

    // Calculate distance from the center
    float fx = x - DIM / 2;
    float fy = y - DIM / 2;
    float d = rsqrtf(fx * fx + fy * fy);  // Use fast inverse square root

    // Compute cosine value for grey level
    float cosine_value = cosf(d / 10.0f - ticks / 7.0f);
    unsigned char grey = (unsigned char)(128.0f + 127.0f * cosine_value / (1.0f / 10.0f * d + 1.0f));

    // Set pixel value in buffer
    int pixel_index = offset * 4;
    ptr[pixel_index + 0] = grey;
    ptr[pixel_index + 1] = grey;
    ptr[pixel_index + 2] = grey;
    ptr[pixel_index + 3] = 255;
}