#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;
#ifndef __HIPCC__
#define __HIPCC__
#endif

#define imin(a, b) (a < b ? a : b)

const int N = 33 * 1024;
const int threadsPerBlock = 256;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void dot(float *a, float *b, float *c)
{
    __shared__ float cache[threadsPerBlock];
    int threadID = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheID = threadIdx.x;
    float temp = 0;

    // Loop unrolling for better performance
    int stepSize = blockDim.x * gridDim.x;
    while (threadID < N) {
        temp += a[threadID] * b[threadID];
        if (threadID + blockDim.x < N) {
            temp += a[threadID + blockDim.x] * b[threadID + blockDim.x];
        }
        if (threadID + 2 * blockDim.x < N) {
            temp += a[threadID + 2 * blockDim.x] * b[threadID + 2 * blockDim.x];
        }
        if (threadID + 3 * blockDim.x < N) {
            temp += a[threadID + 3 * blockDim.x] * b[threadID + 3 * blockDim.x];
        }
        threadID += 4 * stepSize;
    }

    cache[cacheID] = temp;
    __syncthreads();

    // Parallel reduction using loop unrolling
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cacheID < i) {
            cache[cacheID] += cache[cacheID + i];
        }
        __syncthreads();
    }

    if (cacheID == 0) {
        c[blockIdx.x] = cache[0];
    }
}