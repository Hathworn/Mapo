#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c) {
    // Use local variable to reduce global memory access latency
    int result = a + b;
    
    // Efficient single write to global memory
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *c = result;
    }
}