#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int* a, int* b, int* c)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Compute sum if within bounds
    if (id < N)
    {
        c[id] = b[id] + a[id];
    }
}