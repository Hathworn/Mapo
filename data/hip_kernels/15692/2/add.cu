#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int* a, int* b, int* c)
{
int id = threadIdx.x;
if (id < N)
{
c[id] = b[id] + a[id];
}
}