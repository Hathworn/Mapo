#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use a single thread to avoid race conditions
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}