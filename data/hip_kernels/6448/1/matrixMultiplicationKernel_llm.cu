#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define IDX2C(i,j,ld) (((i)*(ld))+(j))

__global__ void matrixMultiplicationKernel(float* A, float* B, float* C, int N)
{
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    float tmpSum = 0.0f;

    if (ROW < N && COL < N) {
        // Optimize using shared memory
        __shared__ float As[16][16]; // Assuming block size of 16x16 or adjust as needed
        __shared__ float Bs[16][16];

        for (int tileIdx = 0; tileIdx < (N + 15) / 16; ++tileIdx) {
            // Load data into shared memory
            if (ROW < N && (tileIdx * 16 + threadIdx.x) < N)
                As[threadIdx.y][threadIdx.x] = A[ROW * N + tileIdx * 16 + threadIdx.x];
            else 
                As[threadIdx.y][threadIdx.x] = 0.0f;

            if (COL < N && (tileIdx * 16 + threadIdx.y) < N)
                Bs[threadIdx.y][threadIdx.x] = B[(tileIdx * 16 + threadIdx.y) * N + COL];
            else
                Bs[threadIdx.y][threadIdx.x] = 0.0f;

            __syncthreads();

            // Compute the partial result
            for (int i = 0; i < 16; ++i)
                tmpSum += As[threadIdx.y][i] * Bs[i][threadIdx.x];

            __syncthreads();
        }

        C[ROW * N + COL] = tmpSum;
    }
}