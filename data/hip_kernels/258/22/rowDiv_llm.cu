#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel
__global__ void rowDiv(float* a, float* b, float* c, int M, int N) {
    int row = blockIdx.x;  // Compute row index
    int col = threadIdx.x;  // Compute column index
    int i = row * blockDim.x + col;
    
    // Ensure index is within bounds
    if (i < M * N) {
        c[i] = a[i] / b[row];  // Perform division with optimized indexing
    }
}