#include "hip/hip_runtime.h"
```cuda
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void colMul(float* a, float* b, float* c, int M, int N) {
    // Calculate row and column indices
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure indices are within bounds
    if (row < M && col < N) {
        int ind = row + col * M; // Compute linear index for matrix `a` and `c`
        c[ind] = a[ind] * b[row]; // Perform element-wise multiplication
    }
}