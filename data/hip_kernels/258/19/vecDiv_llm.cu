#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecDiv(float* a, float* b, float* c, const int N) {
    // Use shared memory to minimize global memory accesses
    __shared__ float sharedA[256]; // Adjust size as needed based on blockDim.x
    __shared__ float sharedB[256]; // Adjust size as needed based on blockDim.x

    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        // Load elements into shared memory
        sharedA[threadIdx.x] = a[i];
        sharedB[threadIdx.x] = b[i];
        __syncthreads();

        // Perform division using shared memory
        c[i] = __fdividef(sharedA[threadIdx.x], sharedB[threadIdx.x]);
    }
}