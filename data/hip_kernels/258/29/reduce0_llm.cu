#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce0(float *g_idata, float *g_odata, int N) {
    extern __shared__ float sdata[];

    // Load elements from global memory to shared memory
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    float sum = 0.0;

    if (i < N) sum = g_idata[i];
    if (i + blockDim.x < N) sum += g_idata[i + blockDim.x];

    sdata[tid] = sum;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Unrolling loop for last warp
    if (tid < 32) {
        volatile float* smem = sdata;  // Use volatile to optimize
        smem[tid] += smem[tid + 32];
        smem[tid] += smem[tid + 16];
        smem[tid] += smem[tid + 8];
        smem[tid] += smem[tid + 4];
        smem[tid] += smem[tid + 2];
        smem[tid] += smem[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}