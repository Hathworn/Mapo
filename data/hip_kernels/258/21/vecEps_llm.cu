#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecEps(float* a, const int N) {
    // Calculate element index and ensure index is within bounds
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        // Check and update element only if it's less than EPS
        if (a[i] < EPS) {
            a[i] = EPS;
        }
    }
}