#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void colMul(float* a, float* b, float* c, int M, int N) {
    // Calculate 2D thread index using block and thread indices
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure within bounds
    if (i < M && j < N) {
        int ind = i + j * M;
        c[ind] = a[ind] * b[i];
    }
}