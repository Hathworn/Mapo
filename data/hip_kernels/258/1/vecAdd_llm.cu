#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(float* __restrict__ a, float* __restrict__ b, float* __restrict__ c, const int N)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop for better performance
    if(i < N) c[i] = a[i] + b[i];
    if(i + blockDim.x < N) c[i + blockDim.x] = a[i + blockDim.x] + b[i + blockDim.x];
}