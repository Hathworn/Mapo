#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void colDiv(float* a, float* b, float* c, int M, int N){

    // Use shared memory to improve memory coalescing
    __shared__ float b_shared[1024];

    // Index for block and thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadIdx.x < M) {
        b_shared[threadIdx.x] = b[threadIdx.x];
    }

    __syncthreads();

    // Ensure grid covers all elements, check i
    if (i < M) {
        int ind = i + blockIdx.y * M;
        c[ind] = a[ind] / b_shared[i];
    }
}