#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecDiv(float* __restrict__ a, float* __restrict__ b, float* __restrict__ c, const int N) {
    // Use shared memory for faster access
    extern __shared__ float shared_b[];
    const int tid = threadIdx.x;
    const int i = blockIdx.x * blockDim.x + tid;

    if (i < N) {
        shared_b[tid] = b[i];  // Load b into shared memory
        __syncthreads();

        // Use shared memory for faster division
        c[i] = __fdividef(a[i], shared_b[tid]);
    }
}