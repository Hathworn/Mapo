#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecMult(float* __restrict__ a, float* __restrict__ b, float* __restrict__ c, const int N) {
    // Use shared memory for performance improvement if necessary
    // Calculate flattened array index
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check bounds to prevent memory access violations
    if(i < N) { 
        c[i] = a[i] * b[i];
    }
}