#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rowDiv(float* a, float* b, float* c, int M, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < M * N) { // Ensure we do not access out of bounds
        c[i] = a[i] / b[blockIdx.x]; // Division operation
    }
}