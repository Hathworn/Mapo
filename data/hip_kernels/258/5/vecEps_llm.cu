#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecEps(float* a, const int N) {
    // Calculate index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a single conditional statement and guard the array access with bound check
    if (i < N && a[i] < EPS)
        a[i] = EPS;
}