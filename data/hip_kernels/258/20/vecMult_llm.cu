#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecMult(float* a, float* b, float* c, const int N) {
    __shared__ float localA[BLOCK_SIZE]; // Use shared memory for a
    __shared__ float localB[BLOCK_SIZE]; // Use shared memory for b
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        // Load data into shared memory
        localA[threadIdx.x] = a[i];
        localB[threadIdx.x] = b[i];
        __syncthreads(); // Ensure all threads have loaded their data

        // Perform the computation
        c[i] = localA[threadIdx.x] * localB[threadIdx.x];
    }
}