#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void colDiv(float* a, float* b, float* c, int M, int N) {
    // Improved index calculation and boundary check
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < M && j < N) {
        int ind = i + j * M;
        c[ind] = a[ind] / b[i];
    }
}