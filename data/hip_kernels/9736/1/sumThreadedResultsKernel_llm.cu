#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumThreadedResultsKernel(long *dev_hist, int *dev_threadedHist, const int valRange, const int Blocks)
{
    int tid = threadIdx.x;
    int stride = blockDim.x * gridDim.x;  // Calculate grid stride

    // Use strided loop for coalesced memory access
    for (int i = tid; i < valRange; i += stride)
    {
        long sum = 0;  // Temporary variable to reduce global memory access
        for (int bl = 0; bl < Blocks; bl++)
        {
            sum += dev_threadedHist[bl * valRange + i];
        }
        dev_hist[i] += sum;
    }
}
```
