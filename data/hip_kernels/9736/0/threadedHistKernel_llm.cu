#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void threadedHistKernel(int *threadedHist, int *arr, const int blockSize, const int valRange, const int threadBlockSize)
{
    int tid = threadIdx.x;
    int pid = blockIdx.x * blockSize + tid;  // Positional ID

    // Use shared memory for reduced access latency
    __shared__ int sharedHist[1024];
    sharedHist[tid] = 0;
    __syncthreads();

    // Each thread increments the relevant position in the local histogram
    for (int i = 0; i < threadBlockSize; i++)
    {
        int val = arr[pid * threadBlockSize + i];
        atomicAdd(&sharedHist[val], 1);
    }
    __syncthreads();

    // Write shared histogram back to global memory
    for (int i = tid; i < valRange; i += blockSize)
    {
        threadedHist[valRange * pid + i] = sharedHist[i];
    }
}