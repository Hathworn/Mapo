#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread indexing for speedup in case accuracy is an array
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx == 0) {
        // Perform division once with the first thread and if there's only one accuracy value
        *accuracy /= N;
    }
}