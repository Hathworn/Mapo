#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Ensure shared memory utilization to optimize division across threads
    __shared__ float shared_accuracy;

    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy;
    }

    __syncthreads();

    // Each thread calculates part of the division
    shared_accuracy /= N;

    __syncthreads();

    if (threadIdx.x == 0) {
        *accuracy = shared_accuracy;
    }
}