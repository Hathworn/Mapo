#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory to reduce global memory access
    __shared__ float shared_accuracy;

    // Initialize shared memory from global memory
    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy;
    }
    __syncthreads();

    // Divide accuracy in shared memory
    if (threadIdx.x == 0) {
        shared_accuracy /= N;
    }
    __syncthreads();

    // Write result back to global memory
    if (threadIdx.x == 0) {
        *accuracy = shared_accuracy;
    }
}