#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Optimization: Ensure accuracy pointer is aligned globally 
    if (threadIdx.x == 0) { // Process only thread 0 for this block
        *accuracy /= N;     // Prevent redundant operations across threads
    }
}