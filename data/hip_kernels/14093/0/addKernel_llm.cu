#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate unique global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if within bounds
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}