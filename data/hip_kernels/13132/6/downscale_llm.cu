#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float *gradInput_data, float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    // Calculate global thread index
    long ii = threadIdx.x + blockDim.x * blockIdx.x;
    ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;

    // Ensure thread is within bounds
    if (ii >= no_elements) return;

    // Optimize nested loops by unrolling
    const int sf_squared = scale_factor * scale_factor;
    for (int sf_iter = 0; sf_iter < sf_squared; ++sf_iter) {
        int i = sf_iter / scale_factor;
        int j = sf_iter % scale_factor;
        
        int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
        gradInput_data[ii] += gradOutput_data[ipidx];
    }
}