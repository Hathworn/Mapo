#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define WARP_SIZE 32

__global__ void cunn_LookupTable_accGradParametersKernel( float *input, float *indices, float *gradOutput, float *gradWeight, float *count, float defaultScale, long numel, long stride) {
    int idx = blockIdx.x * 4 + threadIdx.y;
    
    // Iterate through each input element, avoiding redundant work if ids are the same
    const int SZ = 4;
    
    if (idx < numel && (idx == 0 || input[idx] != input[idx - 1])) {
        do {
            const int startFeature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
            const int weightRow = ((int) input[idx] - 1) * stride;
            const int gradOutputRow = ((int) indices[idx] - 1) * stride;
            const float scale = count ? defaultScale / count[idx] : defaultScale;

            float gradient[SZ];
            float weight[SZ];

            // Load gradOutput and gradWeight into registers
            #pragma unroll
            for (int ii = 0; ii < SZ; ii++) {
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride) {
                    gradient[ii] = gradOutput[gradOutputRow + featureDim];
                    weight[ii] = gradWeight[weightRow + featureDim];
                }
            }
            
            // Compute weight update
            #pragma unroll
            for (int ii = 0; ii < SZ; ii++) {
                weight[ii] += gradient[ii] * scale;
            }

            // Store updated weights back to global memory
            #pragma unroll
            for (int ii = 0; ii < SZ; ii++) {
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride) {
                    gradWeight[weightRow + featureDim] = weight[ii];
                }
            }

            idx++;
        } while (idx < numel && input[idx] == input[idx - 1]);
    }
}