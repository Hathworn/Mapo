#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w)
{
    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // Pre-compute scales to avoid float division inside loops
    float scale_x = float(input_w) / output_w;
    float scale_y = float(input_h) / output_h;

    // Compute gradInput
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        int y_start = static_cast<int>(yy * scale_y);

        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            int x_start = static_cast<int>(xx * scale_x);

            float *ptr_gradInput = gradInput + y_start * input_w + x_start;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float *ptr_ind_x = indices_x + yy * output_w + xx;
            float *ptr_ind_y = indices_y + yy * output_w + xx;

            float z = *ptr_gradOutput;
            int argmax_x = (*ptr_ind_x) - 1;
            int argmax_y = (*ptr_ind_y) - 1;

            // Use atomic operation to prevent race conditions
            atomicAdd(&ptr_gradInput[argmax_x + argmax_y * input_w], z);
        }
    }
}