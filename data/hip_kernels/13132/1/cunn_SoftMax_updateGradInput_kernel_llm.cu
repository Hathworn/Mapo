#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_SoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int nframe, int dim, int stride)
{
    __shared__ float buffer[SOFTMAX_THREADS];
    float *gradInput_k = gradInput + blockIdx.x*dim*stride + blockIdx.y;
    float *output_k = output + blockIdx.x*dim*stride + blockIdx.y;
    float *gradOutput_k = gradOutput + blockIdx.x*dim*stride + blockIdx.y;

    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;

    // Initialize buffer to zero
    float sum = 0;
    for (int i=i_start; i<i_end; i+=i_step)
        sum += gradOutput_k[i*stride] * output_k[i*stride];
    buffer[threadIdx.x] = sum;

    __syncthreads();

    // Reduce using parallel reduction
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            buffer[threadIdx.x] += buffer[threadIdx.x + offset];
        }
        __syncthreads();
    }

    sum = buffer[0];
    for (int i=i_start; i<i_end; i+=i_step)
        gradInput_k[i*stride] = output_k[i*stride] * (gradOutput_k[i*stride] - sum);
}