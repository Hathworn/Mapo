#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Determine block and thread indices
    int timeIdx = threadIdx.x + blockIdx.y * blockDim.x;
    int batchIdx = blockIdx.x;
    
    // Only proceed if within output width bounds
    if (timeIdx < output_w) {
        // Calculate base pointers to gradInput, gradOutput, and indices
        float *gradInput_data = gradInput + batchIdx * input_w * input_n;
        float *gradOutput_data = gradOutput + batchIdx * output_w * input_n + timeIdx * input_n;
        float *indices_data = indices + batchIdx * output_w * input_n + timeIdx * input_n;
        
        // Process each feature
        for (int feat = 0; feat < input_n; ++feat) {
            int index = static_cast<int>(indices_data[feat]);
            atomicAdd(&gradInput_data[index * input_n + feat], gradOutput_data[feat]); // Use atomicAdd for safe updates
        }
    }
}