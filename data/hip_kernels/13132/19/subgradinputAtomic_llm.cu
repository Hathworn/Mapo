#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute global thread indices and other parameters
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;
    int xx = threadIdx.x;
    int yy = blockDim.y * blockIdx.y + threadIdx.y;

    // Select and prepare data for processing
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    float the_weight = weight[k];

    // Check if within output bounds
    if (xx < output_w && yy < output_h) {
        // Calculate pointers
        float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float z = *ptr_gradOutput * the_weight;
        
        // Accumulate results with atomic operations
        for (int ky = 0; ky < kH; ky++) {
            for (int kx = 0; kx < kW; kx++) {
                atomicAdd(&(ptr_gradInput[kx]), z);
            }
            ptr_gradInput += input_w;
        }
    }
}