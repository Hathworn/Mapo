#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradweight(float *input, float *gradOutput, float *gradWeight, float *gradBias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW, float scale)
{
    // Iterators and output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;

    int yy_start = threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    input += i * input_w * input_h;

    // Thread ID
    int tid = blockDim.x * threadIdx.y + threadIdx.x;

    // Create shared memory for partial sums
    extern __shared__ float shared_sums[];
    shared_sums[tid] = 0;

    // Compute partial sums
    for (int yy = yy_start; yy < yy_end; yy += yy_step) {
        for (int xx = xx_start; xx < xx_end; xx += xx_step) {
            float *ptr_input = input + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput;
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    shared_sums[tid] += z * ptr_input[kx];
                }
                ptr_input += input_w;
            }
        }
    }
    __syncthreads();

    // Reduce: accumulate all partial sums to produce final gradWeight
    if (tid == 0) {
        float block_sum = 0;
        for (int i = 0; i < blockDim.x * blockDim.y; i++) {
            block_sum += shared_sums[i];
        }
        atomicAdd(&gradWeight[k], scale * block_sum); // Use atomicAdd for safety
    }
    __syncthreads();

    // Compute gradBias
    shared_sums[tid] = 0;
    for (int i = tid; i < output_w * output_h; i += (blockDim.x * blockDim.y)) {
        shared_sums[tid] += gradOutput[i];
    }
    __syncthreads();

    // Reduce gradBias
    if (tid == 0) {
        float block_sum = 0;
        for (int i = 0; i < (blockDim.x * blockDim.y); i++) {
            block_sum += shared_sums[i];
        }
        atomicAdd(&gradBias[k], scale * block_sum); // Use atomicAdd for safety
    }
}