#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_MSECriterion_updateOutput_kernel(float* output, float *input, float *target, int nframe, int dim, int sizeAverage)
{
    __shared__ float buffer[MSECRITERION_THREADS];
    int k = blockIdx.x;
    float *input_k = input + k*dim;
    float *target_k = target + k*dim;

    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;

    // Initialize buffer to ensure all threads start with zero
    float sum_local = 0.0f;

    // Accumulate mse for each thread over its assigned indices
    for (int i=i_start; i<i_end; i+=i_step)
    {
        float z = input_k[i] - target_k[i];
        sum_local += z*z;
    }
    buffer[threadIdx.x] = sum_local;

    // Synchronize threads to ensure all calculations are complete
    __syncthreads();

    // Perform parallel reduction to compute the final output
    if (threadIdx.x == 0)
    {
        float sum_total = 0.0f;
        for (int i=0; i<blockDim.x; i++)
        {
            sum_total += buffer[i];
        }
        if (sizeAverage)
            sum_total /= dim;
        *output = sum_total;
    }
}