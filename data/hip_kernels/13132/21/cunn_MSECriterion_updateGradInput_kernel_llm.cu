#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory for faster access
__global__ void cunn_MSECriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, float norm, int nframe, int dim)
{
    extern __shared__ float shared_target[];
    
    int k = blockIdx.x;
    float *gradInput_k = gradInput + k * dim;
    float *input_k = input + k * dim;

    int i_start = threadIdx.x;
    int i_step = blockDim.x;

    // Load target data into shared memory
    for (int i = i_start; i < dim; i += i_step) {
        shared_target[i] = target[k * dim + i];
    }
    __syncthreads();

    // Compute gradInput using shared memory
    for (int i = i_start; i < dim; i += i_step) {
        gradInput_k[i] = norm * (input_k[i] - shared_target[i]);
    }
}