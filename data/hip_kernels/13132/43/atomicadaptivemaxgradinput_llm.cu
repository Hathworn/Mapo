#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void atomicadaptivemaxgradinput( float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w )
{
    // Compute global thread indices
    int o = blockIdx.x;
    int i = o;
    int xx = blockIdx.y * blockDim.x + threadIdx.x;
    int yy_increment = blockDim.y * gridDim.y;

    if (xx >= output_w) return; // Ignore threads outside output bounds

    // Offsets for plane selection
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;
    
    // Loop over output height
    for (int yy = blockIdx.y * blockDim.y + threadIdx.y; yy < output_h; yy += yy_increment) {

        int y_start = (int)(yy * input_h / (float)output_h);
        int x_start = (int)(xx * input_w / (float)output_w);

        float *ptr_gradInput = gradInput + y_start * input_w + x_start;
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float *ptr_ind_x = indices_x + yy * output_w + xx;
        float *ptr_ind_y = indices_y + yy * output_w + xx;
        float z = *ptr_gradOutput;

        int argmax_x = (*ptr_ind_x) - 1;
        int argmax_y = (*ptr_ind_y) - 1;

        // Atomic operation ensures safe concurrent updates
        atomicAdd(&(ptr_gradInput[argmax_x + argmax_y * input_w]), z);
    }
}