#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW) {
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int batch_idx = blockIdx.x;
    int k = batch_idx % input_n; // Select input/output plane

    // Calculate Start, End, Step values for xx, yy
    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Point to correct portion of input/output
    float *output_plane = output + batch_idx * output_w * output_h;
    float *input_plane = input + batch_idx * input_w * input_h;

    float the_weight = weight[k];
    float the_bias = bias[k];

    // Loop over output pixels
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            float *input_ptr = input_plane + yy * dH * input_w + xx * dW;
            float *output_ptr = output_plane + yy * output_w + xx;

            float sum = 0.0f;

            // Compute the mean of the input image
            for (int ky = 0; ky < kH; ++ky) {
                for (int kx = 0; kx < kW; ++kx) {
                    sum += input_ptr[kx]; // Accumulate
                }
                input_ptr += input_w; // Move to next row
            }

            // Update output with weight and bias
            *output_ptr = the_weight * sum + the_bias;
        }
    }
}