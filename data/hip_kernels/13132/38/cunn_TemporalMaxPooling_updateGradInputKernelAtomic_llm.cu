#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate total thread index for output
    int index = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    
    // Check if the thread is within bounds
    if (index < output_w) {
        // Calculate per-thread position based offsets
        int gradInput_offset = blockIdx.x * input_w * input_n + index * input_n * dW;
        int gradOutput_offset = blockIdx.x * output_w * input_n + index * input_n;
        int indices_offset = blockIdx.x * output_w * input_n + index * input_n;
        
        // Iterate over all features and perform atomic addition
        for (int feat = 0; feat < input_n; ++feat) {
            int input_idx = static_cast<int>(indices[indices_offset + feat]) * input_n + feat;
            atomicAdd(&gradInput[gradInput_offset + input_idx], gradOutput[gradOutput_offset + feat]);
        }
    }
}