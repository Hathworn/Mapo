#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void krotation_Ax(double *x, double *y, double *z, double xMax, double yMax, double zMax, double omegaX, double omegaY, double omegaZ, double omega, double fudge, double *A) {
    // Obtain global thread index
    int gid = getGid3d3d();
    
    // Compute linear index based on y-dimension of the block and grid
    int yid = blockIdx.y * blockDim.y + threadIdx.y;

    // Optimization: Pre-calculate common expression used in the assignment
    double common_term = -omega * omegaX;

    // Perform the operation
    A[gid] = y[yid] * common_term;
}