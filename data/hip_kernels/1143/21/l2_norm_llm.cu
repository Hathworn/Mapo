#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void l2_norm(double2 *in1, double2 *in2, double2 *in3, double *out) {
    int gid = getGid3d3d();

    // Optimize by reducing redundant calculations
    double x1 = in1[gid].x, y1 = in1[gid].y;
    double x2 = in2[gid].x, y2 = in2[gid].y;
    double x3 = in3[gid].x, y3 = in3[gid].y;
    
    // Precompute squares
    double square1 = x1 * x1 + y1 * y1;
    double square2 = x2 * x2 + y2 * y2;
    double square3 = x3 * x3 + y3 * y3;
    
    // Compute normalized sum
    out[gid] = sqrt(square1 + square2 + square3);
}