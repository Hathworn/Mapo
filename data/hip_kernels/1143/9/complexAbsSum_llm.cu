#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.y * blockDim.x) + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void complexAbsSum(double2 *in1, double2 *in2, double2 *in3, double *out){
    // Compute global index using HIP built-in functions for improved readability
    int gid = getGid3d3d();
    // Preload input values for better memory access efficiency
    double2 val1 = in1[gid];
    double2 val2 = in2[gid];
    double2 val3 = in3[gid];
    
    // Accumulate values with fewer operations
    double sumX = val1.x + val2.x + val3.x;
    double sumY = val1.y + val2.y + val3.y;
    
    out[gid] = sqrt(sumX * sumX + sumY * sumY);
}