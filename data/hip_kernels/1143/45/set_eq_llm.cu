#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.z * blockDim.y * blockDim.x)
                 + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__global__ void set_eq(double *in1, double *in2) {
    // Calculate global thread ID
    int gid = getGid3d3d();
    // Copy data from in1 to in2
    in2[gid] = in1[gid];
}