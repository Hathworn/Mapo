#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double complexMagnitude(double2 in){
    return sqrt(in.x*in.x + in.y*in.y);
}

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.z * blockDim.y * blockDim.x) // Reordered for better memory access pattern
                 + (threadIdx.y * blockDim.x)
                 + threadIdx.x;
    return threadId;
}

__global__ void complexMagnitudeSquared(double2 *in, double2 *out) {
    int gid = getGid3d3d();
    if(gid < gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z) { // Global boundary check
        double2 val = in[gid];
        out[gid].x = val.x * val.x + val.y * val.y; // Localize memory access
        out[gid].y = 0;
    }
}