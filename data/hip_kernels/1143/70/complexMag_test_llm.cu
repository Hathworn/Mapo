#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double complexMagnitude(double2 in) {
    return sqrt(in.x * in.x + in.y * in.y);
}

__global__ void complexMag_test(double2 *in, double *out) {
    // Use block and thread indices for efficient parallel processing
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure that only valid indices process data
    if (idx < 1) {
        out[idx] = complexMagnitude(in[idx]);
    }
}