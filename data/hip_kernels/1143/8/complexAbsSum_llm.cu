#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void complexAbsSum(double2 *in1, double2 *in2, double *out) {
    int gid = getGid3d3d();
    
    // Unroll computation of temp
    double real = in1[gid].x + in2[gid].x;
    double imag = in1[gid].y + in2[gid].y;
    
    // Direct computation without temporary storage
    out[gid] = sqrt(real * real + imag * imag);
}