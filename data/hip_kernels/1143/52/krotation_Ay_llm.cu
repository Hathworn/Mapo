#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.z * blockDim.y * blockDim.x) 
                 + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__global__ void krotation_Ay(double *x, double *y, double *z, double xMax, double yMax, double zMax, 
                             double omegaX, double omegaY, double omegaZ, double omega, double fudge, double *A) {
    int gid = getGid3d3d();
    if (gid < gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z) {  // Ensure gid is within bounds
        int xid = blockDim.x * blockIdx.x + threadIdx.x;  // Calculate X index
        A[gid] = x[xid] * omega * omegaY;  // Compute result
    }
}