#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                    + threadIdx.x 
                    + threadIdx.y * blockDim.x 
                    + threadIdx.z * blockDim.x * blockDim.y;
    return threadId;
}

__global__ void pSum(double* in1, double* output, int pass){
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x;
    unsigned int gid = getGid3d3d(); // Global ID assignment
    extern __shared__ double sdata2[];

    // Load input data into shared memory
    sdata2[tid] = in1[gid];
    __syncthreads();

    // Reduce using the shared memory
    for(int i = blockDim.x >> 1; i > 0; i >>= 1){
        if(tid < i){ // Correction in conditional check
            sdata2[tid] += sdata2[tid + i];
        }
        __syncthreads();
    }

    if(tid == 0){
        output[bid] = sdata2[0]; // Write result for this block to output
    }
}