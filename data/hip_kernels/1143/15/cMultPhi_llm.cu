#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) 
                 + (threadIdx.y * blockDim.x) 
                 + threadIdx.x;
    return threadId;
}

// Optimized global kernel function
__global__ void cMultPhi(double2* in1, double* in2, double2* out) {
    unsigned int gid = getGid3d3d();
    
    double2 in1Val = in1[gid]; // Cache in1[gid]
    double in2Val = in2[gid];  // Cache in2[gid]
    
    // Precompute trigonometric functions
    double cosVal = cos(in2Val);
    double sinVal = sin(in2Val);
    
    // Compute result using precomputed values
    double2 result;
    result.x = cosVal * in1Val.x - in1Val.y * sinVal;
    result.y = in1Val.x * sinVal + in1Val.y * cosVal;
    
    out[gid] = result;
}