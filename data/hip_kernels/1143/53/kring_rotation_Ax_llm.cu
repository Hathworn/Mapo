#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                + threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.x + threadIdx.x;
    return threadId;
}

__global__ void kring_rotation_Ax(double *x, double *y, double *z, double xMax, double yMax, double zMax,
                                  double omegaX, double omegaY, double omegaZ, double omega, double fudge, double *A) {
                                   
    // Calculate unique global thread ID
    int gid = getGid3d3d();
    
    // Calculate thread coordinates in 3D space
    int xid = blockDim.x * blockIdx.x + threadIdx.x;
    int yid = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Precompute values to avoid repeated calculations
    double precomputedZ = z[blockDim.z * blockIdx.z + threadIdx.z] + zMax; // Move computation here
    double theta = atan2(y[yid], x[xid]);
    double cosTheta = cos(theta);

    // Calculate the value for A matrix
    A[gid] = precomputedZ * cosTheta * omega * omegaX;
}