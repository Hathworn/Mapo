#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.z * blockDim.x * blockDim.y) // Moved threadIdx.z earlier for better coalescing
                 + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__global__ void thread_test(double *in, double *out){
    unsigned int Gid = getGid3d3d();
    
    // Use direct store for single transaction. Avoid direct computation to reduce warp divergence.
    out[Gid] = static_cast<double>(Gid);
    // in[Gid] = static_cast<double>(Gid); // Comment preserved for potential future use
}