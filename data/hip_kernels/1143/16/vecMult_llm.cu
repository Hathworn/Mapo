#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void vecMult(double2 *in, double *factor, double2 *out) {
    // Calculate unique global thread ID
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x
                     + (blockDim.x * gridDim.x) * (blockIdx.y + blockIdx.z * gridDim.y);

    // Ensure thread is within bounds
    if (gid < gridDim.x * blockDim.x * gridDim.y * gridDim.z * blockDim.y * blockDim.z) {
        // Perform multiplication and store result
        out[gid].x = in[gid].x * factor[gid];
        out[gid].y = in[gid].y * factor[gid];
    }
}