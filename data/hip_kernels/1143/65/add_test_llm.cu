#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double2 add(double2 a, double2 b) {
    return {a.x + b.x, a.y + b.y};
}

__global__ void add_test(double2 *a, double2 *b, double2 *c) {
    // Use threadIdx and blockIdx for parallel computing
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    c[idx] = add(a[idx], b[idx]);
}