#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) +
                   (threadIdx.y * blockDim.x) +
                   (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

// Optimized kernel function for better memory throughput
__global__ void make_cufftDoubleComplex(double *in, double2 *out) {
    int gid = getGid3d3d();
    // Ensure the thread is within array bounds
    if (gid < gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z) {
        double value = in[gid]; // Avoid redundant memory access
        out[gid] = make_double2(value, 0.0); // Simplified assignment using make_double2
    }
}