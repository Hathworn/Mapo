#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    // Simplified global index calculation
    return threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * (threadIdx.z + blockDim.z * (blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z))));
}

__global__ void vecSum(double *in, double *factor, double *out) {
    unsigned int gid = getGid3d3d();
    // Combine operations to reduce overhead
    out[gid] = in[gid] + factor[gid];
}