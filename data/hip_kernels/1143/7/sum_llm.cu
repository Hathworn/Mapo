#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define THREADS_PER_BLOCK 256

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) 
                 + (threadIdx.z * blockDim.y * blockDim.x) 
                 + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__global__ void sum(double2 *in1, double2 *in2, double2 *out) {
    // Optimize by computing gid using built-in functionality
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x + 
                       blockIdx.y * blockDim.x * gridDim.x + 
                       blockIdx.z * gridDim.x * gridDim.y * blockDim.x;
    if (gid < gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z) {
        out[gid].x = in1[gid].x + in2[gid].x;
        out[gid].y = in1[gid].y + in2[gid].y;
    }
}