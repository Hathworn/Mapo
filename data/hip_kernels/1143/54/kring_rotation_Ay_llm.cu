#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void kring_rotation_Ay(double *x, double *y, double *z, double xMax, double yMax, double zMax, double omegaX, double omegaY, double omegaZ, double omega, double fudge, double *A) {
    // Utilize shared memory for frequently accessed data
    __shared__ double sx[BLOCK_SIZE];
    __shared__ double sy[BLOCK_SIZE];
    
    int xid = blockDim.x * blockIdx.x + threadIdx.x;
    int yid = blockDim.y * blockIdx.y + threadIdx.y;
    int zid = blockDim.z * blockIdx.z + threadIdx.z;
    int gid = getGid3d3d();

    // Load data into shared memory
    if (threadIdx.x < blockDim.x) sx[threadIdx.x] = x[xid];
    if (threadIdx.y < blockDim.y) sy[threadIdx.y] = y[yid];
    __syncthreads();
    
    // Ensure thread safety and avoid race conditions
    if(xid < xMax && yid < yMax && zid < zMax) {
        // Compute only after ensuring data is available in shared memory
        double theta = atan2(sy[threadIdx.y], sx[threadIdx.x]);
        A[gid] = (z[zid] + zMax) * sin(theta) * omega * omegaX;
    }
}