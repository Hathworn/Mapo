#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
               + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void angularOp(double omega, double dt, double2* wfc, double* xpyypx, double2* out) {
    unsigned int gid = getGid3d3d();

    // Use local registers for temporary storage to reduce repeated memory access
    double2 wfc_val = wfc[gid];
    double xpyypx_val = xpyypx[gid];

    // Precompute common repetitive calculation
    double op = exp(-omega * xpyypx_val * dt);

    // Perform calculations using the precomputed value
    double2 result;
    result.x = wfc_val.x * op;
    result.y = wfc_val.y * op;

    // Output result to global memory
    out[gid] = result;
}