#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) 
                 + (threadIdx.z * blockDim.y * blockDim.x) 
                 + (threadIdx.y * blockDim.x) 
                 + threadIdx.x;
    return threadId;
}

__global__ void aux_fields(double *V, double *K, double gdt, double dt, double* Ax, double *Ay, double* Az, double *px, double *py, double *pz, double* pAx, double* pAy, double* pAz, double2* GV, double2* EV, double2* GK, double2* EK, double2* GpAx, double2* GpAy, double2* GpAz, double2* EpAx, double2* EpAy, double2* EpAz) {
    int gid = getGid3d3d();
    int xid = blockDim.x * blockIdx.x + threadIdx.x;
    int yid = blockDim.y * blockIdx.y + threadIdx.y;
    int zid = blockDim.z * blockIdx.z + threadIdx.z;

    double v_factor = V[gid] * (gdt / (2 * HBAR));
    double k_factor = K[gid] * (gdt / HBAR);
    GV[gid] = make_double2(exp(-v_factor), 0.0); // Combine initialization
    GK[gid] = make_double2(exp(-k_factor), 0.0); 

    // Precompute for magnetic field calculations
    // Use registers to reduce memory access latency
    double px_val = Ax[gid] * px[xid];
    double py_val = Ay[gid] * py[yid];
    double pz_val = Az[gid] * pz[zid];
    
    pAx[gid] = px_val;
    pAy[gid] = py_val;
    pAz[gid] = pz_val;

    GpAx[gid] = make_double2(exp(-px_val * gdt), 0.0);
    GpAy[gid] = make_double2(exp(-py_val * gdt), 0.0);
    GpAz[gid] = make_double2(exp(-pz_val * gdt), 0.0);

    double ev_factor = V[gid] * (dt / (2 * HBAR));
    double ek_factor = K[gid] * (dt / HBAR);
    
    EV[gid].x = cos(-ev_factor); // Precompute sine and cosine
    EV[gid].y = sin(-ev_factor);
    EK[gid].x = cos(-ek_factor);
    EK[gid].y = sin(-ek_factor);

    EpAz[gid].x = cos(-pz_val * dt);
    EpAz[gid].y = sin(-pz_val * dt);
    EpAy[gid].x = cos(-py_val * dt);
    EpAy[gid].y = sin(-py_val * dt);
    EpAx[gid].x = cos(-px_val * dt);
    EpAx[gid].y = sin(-px_val * dt);
}