#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void l2_norm(double *in1, double *in2, double *in3, double *out) {
    // Get unique thread Id for current thread
    int gid = getGid3d3d();
    
    // Load values to local variables to reduce global memory access
    double val1 = in1[gid];
    double val2 = in2[gid];
    double val3 = in3[gid];
    
    // Calculate L2 norm using local variables
    out[gid] = sqrt(val1 * val1 + val2 * val2 + val3 * val3);
}