#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
    + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
    + (threadIdx.y * blockDim.x)
    + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void kring_rotation_Az(double *x, double *y, double *z, double xMax, double yMax, double zMax, double omegaX, double omegaY, double omegaZ, double omega, double fudge, double *A){
    int gid = getGid3d3d();
    int xid = blockDim.x * blockIdx.x + threadIdx.x;
    int yid = blockDim.y * blockIdx.y + threadIdx.y;

    // Cache x[xid] and y[yid] values to avoid redundant global memory accesses
    double x_val = x[xid];
    double y_val = y[yid];

    // Compute radius using cached values
    double r = sqrt(x_val * x_val + y_val * y_val);

    // Calculate and store result
    A[gid] = r * omega * omegaX;
}