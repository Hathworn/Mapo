#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    // Optimize block and thread indices calculation.
    int blockId = blockIdx.z * gridDim.x * gridDim.y + blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
    return blockId * (blockDim.x * blockDim.y * blockDim.z) + threadId;
}

__global__ void scalarDiv_wfcNorm(double2* in, double dr, double* pSum, double2* out) {
    // Use shared memory for norm calculation
    __shared__ double sharedNorm;

    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
        sharedNorm = sqrt((pSum[0]) * dr);
    }
    
    __syncthreads(); // Ensure all threads have the loaded norm

    unsigned int gid = getGid3d3d();
    double2 result;
    double norm = sharedNorm;
    result.x = (in[gid].x / norm);
    result.y = (in[gid].y / norm);
    out[gid] = result;
}