#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define HBAR 1.0545718e-34 // Define Planck constant if not already defined

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void simple_K(double *xp, double *yp, double *zp, double mass, double *K) {
    unsigned int gid = getGid3d3d();

    // Pre-compute HBAR^2 / (2 * mass) for reuse
    double factor = HBAR * HBAR / (2 * mass);

    // Use single index calculations for efficiency
    unsigned int xid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yid = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int zid = blockIdx.z * blockDim.z + threadIdx.z;

    // Ensure indices are within bounds
    if (xid < gridDim.x * blockDim.x && yid < gridDim.y * blockDim.y && zid < gridDim.z * blockDim.z) {
        double x_val = xp[xid];
        double y_val = yp[yid];
        double z_val = zp[zid];

        // Compute energy contribution from x, y, z components
        K[gid] = factor * (x_val * x_val + y_val * y_val + z_val * z_val);
    }
}