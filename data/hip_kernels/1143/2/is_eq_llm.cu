#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.y * blockDim.x) + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void is_eq(bool *a, bool *b, bool *ans) {
    // Get unique global thread index
    int gid = getGid3d3d();

    // Use shared memory for comparison result
    __shared__ bool isEqualShared;

    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
        isEqualShared = true; // Initialize shared memory
    }
    __syncthreads();

    if (a[gid] != b[gid]) {
        isEqualShared = false; // Update shared memory on mismatch
    }
    __syncthreads();

    // Only first thread writes the result back to global memory
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
        ans[0] = isEqualShared;
    }
}