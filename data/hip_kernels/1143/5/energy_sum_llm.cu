#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + threadIdx.z * (blockDim.x * blockDim.y)
                 + threadIdx.y * blockDim.x
                 + threadIdx.x;
    return threadId;
}

__global__ void energy_sum(double2 *in1, double2 *in2, double *out) {
    // Use shared memory for better memory access patterns
    __shared__ double2 shared_in1[blockDim.x * blockDim.y * blockDim.z];
    __shared__ double2 shared_in2[blockDim.x * blockDim.y * blockDim.z];

    int local_tid = threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
    int global_tid = getGid3d3d();

    // Load data into shared memory
    shared_in1[local_tid] = in1[global_tid];
    shared_in2[local_tid] = in2[global_tid];
    __syncthreads();

    // Perform computation using shared memory to reduce global memory accesses
    out[global_tid] = shared_in1[local_tid].x + shared_in2[local_tid].x;
}