#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double complexMagnitude(double2 in){
    return sqrt(in.x * in.x + in.y * in.y);
}

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) +
                   (threadIdx.y * blockDim.x) +
                   (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void complexMagnitude(double2 *in, double *out){
    // Use shared memory for faster access
    __shared__ double2 sharedIn[256]; // Assuming blockDim.x * blockDim.y * blockDim.z <= 256

    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    int gid = getGid3d3d();

    // Load data into shared memory
    sharedIn[tid] = in[gid];
    __syncthreads();

    // Compute magnitude using shared memory
    if (gid < gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z) { // Check boundary condition
        out[gid] = sqrt(sharedIn[tid].x * sharedIn[tid].x + sharedIn[tid].y * sharedIn[tid].y);
    }
}