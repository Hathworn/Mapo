#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__device__ double2 pow(double2 a, int b){
    double r = sqrt(a.x*a.x + a.y*a.y);
    double theta = atan(a.y / a.x);
    return {pow(r, b) * cos(b * theta), pow(r, b) * sin(b * theta)};
}

__global__ void scalarPow(double2* in, double param, double2* out){
    // Calculate global ID
    unsigned int gid = getGid3d3d();
    
    // Using temporary variables to avoid multiple indexing
    double2 input = in[gid];
    
    // Compute powers efficiently; avoid redundant pow function calls
    double r = sqrt(input.x * input.x + input.y * input.y);
    double theta = atan2(input.y, input.x); // Improved precision with atan2
    double pow_r_param = pow(r, param);
    
    // Store result directly after computation
    out[gid] = {pow_r_param * cos(param * theta), pow_r_param * sin(param * theta)};
}