#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void energy_lsum(double *in1, double2 *in2, double *out){
    // Calculate global thread ID more efficiently using a function
    int gid = getGid3d3d();

    // Use shared memory to reduce global memory accesses
    extern __shared__ double sharedData[];
    sharedData[threadIdx.x] = in1[gid] + in2[gid].x;
    __syncthreads();

    // Write result back to global memory
    out[gid] = sharedData[threadIdx.x];
}