#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
    return threadId;
}

__device__ double2 pow(double2 a, int b){
    double r = sqrt(a.x * a.x + a.y * a.y);
    double theta = atan2(a.y, a.x);  // Use atan2 for more robustness 
    return {pow(r, b) * cos(b * theta), pow(r, b) * sin(b * theta)};
}

__global__ void ktorus_wfc(double *x, double *y, double *z, double *items, double winding, double *phi, double2 *wfc){
    int gid = getGid3d3d();
    int xid = blockIdx.x * blockDim.x + threadIdx.x;
    int yid = blockIdx.y * blockDim.y + threadIdx.y;
    int zid = blockIdx.z * blockDim.z + threadIdx.z;

    if (xid < gridDim.x * blockDim.x && yid < gridDim.y * blockDim.y && zid < gridDim.z * blockDim.z) {
        double rad = sqrt((x[xid] - items[6]) * (x[xid] - items[6]) + (y[yid] - items[7]) * (y[yid] - items[7])) - 0.5 * items[0];

        double expArg = pow(rad / (items[14] * items[15] * 0.5), 2) + pow(z[zid] / (items[14] * items[17] * 0.5), 2);
        wfc[gid].x = exp(-expArg);
        wfc[gid].y = 0.0;
    }
}