#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + threadIdx.z * (blockDim.x * blockDim.y)
                 + threadIdx.y * blockDim.x
                 + threadIdx.x;
    return threadId;
}

__global__ void derive(double2 *data, double2 *out, int stride, int gsize, double dx) {
    int gid = getGid3d3d();
    // Ensure gid is within bounds
    if (gid < gsize) {
        double2 result;
        if (gid + stride < gsize) {
            // Compute derivative using stride
            result.x = (data[gid + stride].x - data[gid].x) / dx;
            result.y = (data[gid + stride].y - data[gid].y) / dx;
        } else {
            // Handle boundary condition
            result.x = data[gid].x / dx;
            result.y = data[gid].y / dx;
        }
        // Write result to output
        out[gid] = result;
    }
}