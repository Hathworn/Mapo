#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 256 // Define block size for optimization

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                  + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                   + (threadIdx.y * blockDim.x)
                   + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void l2_norm(double *in1, double *in2, double *out) {
    // Calculate global thread ID
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread ID is within array bounds
    if (gid < gridDim.x * blockDim.x * blockDim.y * blockDim.z) {
        // Compute square root of sum of squares
        double val1 = in1[gid];
        double val2 = in2[gid];
        out[gid] = sqrt(val1 * val1 + val2 * val2);
    }
}