#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) 
                   + (threadIdx.z * blockDim.y * blockDim.x)  // Optimize thread index calculation
                   + (threadIdx.y * blockDim.x) + threadIdx.x; 
    return threadId;
}

__global__ void kstd_wfc(double *x, double *y, double *z, double *items, double winding, double *phi, double2 *wfc) {

    int gid = getGid3d3d();
    int xid = blockDim.x * blockIdx.x + threadIdx.x;
    int yid = blockDim.y * blockIdx.y + threadIdx.y;
    int zid = blockDim.z * blockIdx.z + threadIdx.z;

    double invItems14Squared = 1.0 / (items[14] * items[14]);  // Pre-compute inverse and square
    double expFactor = exp(-(x[xid] * x[xid] * invItems14Squared / (items[15] * items[15]) 
                         + y[yid] * y[yid] * invItems14Squared / (items[16] * items[16])
                         + z[zid] * z[zid] * invItems14Squared / (items[17] * items[17])));

    phi[gid] = -fmod(winding * atan2(y[yid], x[xid]), 2 * PI);

    double cosPhi = cos(phi[gid]);
    double sinPhi = sin(phi[gid]);

    // Minimize repeated computations
    wfc[gid].x = expFactor * cosPhi;
    wfc[gid].y = -expFactor * sinPhi;
}