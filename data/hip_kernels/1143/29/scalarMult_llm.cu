#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void scalarMult(double* in, double factor, double* out){
    unsigned int gid = getGid3d3d();
    // Load input element, compute the result, and store in output array
    double value = in[gid];
    out[gid] = value * factor;
}