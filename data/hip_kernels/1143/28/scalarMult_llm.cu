#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                   + (threadIdx.y * blockDim.x)
                   + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void scalarMult(double2* in, double factor, double2* out){
    // Calculate global index efficiently using built-in variables without extra function call
    unsigned int gid = getGid3d3d();
    
    // Load input in fewer operations
    double2 input_val = in[gid];
    
    // Directly calculate and store the result to reduce register pressure
    out[gid].x = input_val.x * factor;
    out[gid].y = input_val.y * factor;
}