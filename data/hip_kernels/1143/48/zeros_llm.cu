#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void zeros(double *field, int n) {
    int xid = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Use threadIdx.x < blockDim.x to ensure valid memory access
    if (xid < n) {
        field[xid] = 0;
    }
}