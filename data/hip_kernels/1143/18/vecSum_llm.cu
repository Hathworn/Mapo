#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
        + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
        + (threadIdx.y * blockDim.x)
        + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}
__global__ void vecSum(double2 *in, double *factor, double2 *out){
    unsigned int gid = getGid3d3d();
    // Load input values into registers to optimize memory access
    double2 in_val = in[gid];
    double factor_val = factor[gid];
    double2 result;
    // Perform computation using registers
    result.x = in_val.x + factor_val;
    result.y = in_val.y + factor_val;
    // Store result back to global memory
    out[gid] = result;
}