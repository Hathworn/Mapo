#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getTid3d3d() {
    return blockDim.x * (blockDim.y * threadIdx.z + threadIdx.y) + threadIdx.x;
}

__device__ unsigned int getBid3d3d() {
    return blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z);
}

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    return blockId * (blockDim.x * blockDim.y * blockDim.z) +
           (threadIdx.z * blockDim.x * blockDim.y) +
           (threadIdx.y * blockDim.x) +
           threadIdx.x;
}

__device__ double2 mult(double2 a, double2 b) {
    return {a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x};
}

__device__ double2 mult(double2 a, double b) {
    return {a.x * b, a.y * b};
}

__global__ void multipass(double* input, double* output) {
    unsigned int tid = getTid3d3d();  // Simplify thread index calculation
    unsigned int gid = getGid3d3d();  // Simplify global index calculation
    
    extern __shared__ double sdatad[];
    
    sdatad[tid] = input[gid];  // Load input into shared memory
    __syncthreads();
    
    for(int i = blockDim.x >> 1; i > 0; i >>= 1) {
        if(tid < i) {
            sdatad[tid] += sdatad[tid + i];  // Reduce in shared memory
        }
        __syncthreads();
    }
    
    if(tid == 0) {
        output[getBid3d3d()] = sdatad[0];  // Write result from block to output
    }
}