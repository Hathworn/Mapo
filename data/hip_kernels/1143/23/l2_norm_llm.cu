#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
    + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
    + (threadIdx.y * blockDim.x)
    + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void l2_norm(double2 *in1, double2 *in2, double *out){
    int gid = getGid3d3d();

    // Use temporary variables to store components
    double x1 = in1[gid].x;
    double y1 = in1[gid].y;
    double x2 = in2[gid].x;
    double y2 = in2[gid].y;

    // Calculate the squared magnitudes more clearly
    double norm_val = x1 * x1 + y1 * y1 + x2 * x2 + y2 * y2;

    // Store the result
    out[gid] = sqrt(norm_val);
}