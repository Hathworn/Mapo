#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ unsigned int getGid3d3d() {
    return blockIdx.x + blockIdx.y * gridDim.x
        + gridDim.x * gridDim.y * blockIdx.z;
}

__global__ void cMult(double2* in1, double2* in2, double2* out) {
    unsigned int gid = getGid3d3d() * blockDim.x * blockDim.y * blockDim.z
                       + threadIdx.z * blockDim.x * blockDim.y
                       + threadIdx.y * blockDim.x + threadIdx.x;
                       
    double2 tin1 = in1[gid];
    double2 tin2 = in2[gid];
    
    // Perform complex number multiplication
    out[gid].x = tin1.x * tin2.x - tin1.y * tin2.y;
    out[gid].y = tin1.x * tin2.y + tin1.y * tin2.x;
}