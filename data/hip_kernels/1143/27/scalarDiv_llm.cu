#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Calculate unique thread ID for 3D grid and blocks
__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + threadIdx.z * (blockDim.x * blockDim.y)
                 + threadIdx.y * blockDim.x
                 + threadIdx.x;
    return threadId;
}

// Optimized kernel function for scalar division
__global__ void scalarDiv(double* __restrict__ in, double factor, double* __restrict__ out) {
    unsigned int gid = getGid3d3d();

    // Ensure gid does not go out of bounds
    if(gid < gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z) {
        out[gid] = in[gid] / factor;
    }
}