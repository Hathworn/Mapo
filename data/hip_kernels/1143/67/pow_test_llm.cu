#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// __device__ function to calculate power of complex number using polar coordinates
__device__ double2 pow(double2 a, int b) {
    double r = sqrt(a.x*a.x + a.y*a.y);  // Compute magnitude
    double theta = atan2(a.y, a.x);      // Compute angle using atan2 for improved accuracy
    double powered_r = pow(r, b);        // Calculate r raised to power b
    double b_theta = b * theta;          // Calculate b times theta
    return {powered_r * cos(b_theta), powered_r * sin(b_theta)};
}

// Optimized __global__ kernel function
__global__ void pow_test(double2 *a, int b, double2 *c) {
    // Use threadIdx.x and blockIdx to support parallel execution
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    c[index] = pow(a[index], b);         // Assign result using parallel indexing
}