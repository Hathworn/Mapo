#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                  + threadIdx.z * (blockDim.x * blockDim.y)
                  + threadIdx.y * blockDim.x
                  + threadIdx.x;
    return threadId;
}

__global__ void ktest_Ax(double *x, double *y, double *z, double xMax, double yMax, double zMax, double omegaX, double omegaY, double omegaZ, double omega, double fudge, double *A) {
    int gid = getGid3d3d();
    
    // Precompute values shared by threads in the same block
    __shared__ double shared_yMaxOmega;
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
        shared_yMaxOmega = yMax * omega;
    }
    __syncthreads(); // Ensure all threads have the shared value

    int yid = blockDim.y * blockIdx.y + threadIdx.y;
    A[gid] = (sin(y[yid] * 100000) + 1) * shared_yMaxOmega;  // Use shared value
}