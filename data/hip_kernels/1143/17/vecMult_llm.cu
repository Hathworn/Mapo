#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void vecMult(double *in, double *factor, double *out) {
    // Calculate the global thread ID
    unsigned int gid = getGid3d3d();
    
    // Check if thread ID is within array bounds for safety
    if (gid < (gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z)) {
        // Perform the vector multiplication
        double result = in[gid] * factor[gid];

        // Store the result
        out[gid] = result;
    }
}