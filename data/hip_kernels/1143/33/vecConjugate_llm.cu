#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
        + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
        + (threadIdx.y * blockDim.x)
        + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void vecConjugate(double2 *in, double2 *out){
    // Calculate 3D thread ID
    unsigned int gid = getGid3d3d();
    
    // Ensure the thread is within bounds
    if (gid < (gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z)) {
        // Read input and compute conjugate
        double2 result;
        result.x = in[gid].x;
        result.y = -in[gid].y;
    
        // Write result to output
        out[gid] = result;
    }
}