#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void derive(double *data, double *out, int stride, int gsize, double dx){
    int gid = getGid3d3d();
    if (gid >= gsize) return;  // Early exit for threads with gid >= gsize

    double result = data[gid] / dx;  // Calculate initial result

    if (gid + stride < gsize) {
        result = (data[gid + stride] - data[gid]) / dx; // Update result when gid + stride < gsize
    }

    out[gid] = result;  // Store computed result in out array
}