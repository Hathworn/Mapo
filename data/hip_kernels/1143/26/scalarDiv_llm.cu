#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + threadIdx.x 
                 + threadIdx.y * blockDim.x 
                 + threadIdx.z * (blockDim.x * blockDim.y);
    return threadId;
}

__global__ void scalarDiv(double2* in, double factor, double2* out){
    unsigned int gid = getGid3d3d();
    if (gid < gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z) {
        // Check if gid is within bounds
        double2 result;
        result.x = in[gid].x / factor;  // Perform division for x
        result.y = in[gid].y / factor;  // Perform division for y
        out[gid] = result;  // Store result in output array
    }
}