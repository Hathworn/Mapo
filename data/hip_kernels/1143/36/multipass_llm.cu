#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getTid3d3d(){
    return threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z);
}

__device__ unsigned int getBid3d3d(){
    return blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z);
}

__device__ unsigned int getGid3d3d(){
    return (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
}

__device__ double2 mult(double2 a, double2 b){
    return {a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x};
}

__device__ double2 mult(double2 a, double b){
    return {a.x * b, a.y * b};
}

__global__ void multipass(double2* input, double2* output, int pass){
    unsigned int tid = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z);
    unsigned int bid = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
    unsigned int gid = tid + blockDim.x * blockDim.y * blockDim.z * bid;
    
    extern __shared__ double2 sdata[];
    sdata[tid] = input[gid];
    __syncthreads();
    
    for(int i = blockDim.x >> 1; i > 0; i >>= 1){
        if(tid < i){
            sdata[tid].x += sdata[tid + i].x;  // Reduce x components
            sdata[tid].y += sdata[tid + i].y;  // Reduce y components
        }
        __syncthreads();
    }
    
    if(tid == 0){
        output[bid] = sdata[0];  // Store the result
    }
}