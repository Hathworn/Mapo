#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void kring_Az(double *x, double *y, double *z, double xMax, double yMax, double zMax, double omegaX, double omegaY, double omegaZ, double omega, double fudge, double *A){
    int gid = getGid3d3d();

    // Optimize these calculations to avoid unnecessary reads and recompute
    int xid = blockIdx.x * blockDim.x + threadIdx.x;
    int yid = blockIdx.y * blockDim.y + threadIdx.y;

    double x_val = x[xid]; // Cache global memory read
    double y_val = y[yid]; // Cache global memory read
    double rad = sqrt(x_val * x_val + y_val * y_val);

    // Pre-compute constant expression
    double exp_factor = __expf(-rad * rad / (0.0001f * xMax)) * 0.01f;

    A[gid] = omega * exp_factor;
}