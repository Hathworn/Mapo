#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void kconstant_A(double *x, double *y, double *z, double xMax, double yMax, double zMax, double omegaX, double omegaY, double omegaZ, double omega, double fudge, double *A) {
    // Utilize blockIdx and threadIdx to directly compute gid for increased performance
    int gid = blockIdx.z * (gridDim.y * gridDim.x * blockDim.x * blockDim.y * blockDim.z)
            + blockIdx.y * (gridDim.x * blockDim.x * blockDim.y * blockDim.z)
            + blockIdx.x * (blockDim.x * blockDim.y * blockDim.z)
            + threadIdx.z * (blockDim.x * blockDim.y)
            + threadIdx.y * blockDim.x
            + threadIdx.x;
    
    // Direct memory access for better optimization
    A[gid] = 0.0;
}