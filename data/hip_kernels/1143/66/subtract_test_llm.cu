#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double2 subtract(double2 a, double2 b) {
    return {a.x - b.x, a.y - b.y};
}

__global__ void subtract_test(double2 *a, double2 *b, double2 *c) {
    // Use threadIdx.x assuming only processing one element in this example
    int idx = threadIdx.x;
    c[idx] = subtract(a[idx], b[idx]);
}