#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void kharmonic_V(double *x, double *y, double *z, double* items, double *Ax, double *Ay, double *Az, double *V) {
    int gid = getGid3d3d();

    // Use shared memory to load items data to reduce global memory access
    __shared__ double s_items[12];
    if (threadIdx.x < 12) {
        s_items[threadIdx.x] = items[threadIdx.x];
    }
    __syncthreads();

    // Calculate indices
    int xid = blockDim.x * blockIdx.x + threadIdx.x;
    int yid = blockDim.y * blockIdx.y + threadIdx.y;
    int zid = blockDim.z * blockIdx.z + threadIdx.z;

    // Compute potentials
    double V_x = s_items[3] * (x[xid] + s_items[6]);
    double V_y = s_items[10] * s_items[4] * (y[yid] + s_items[7]);
    double V_z = s_items[11] * s_items[5] * (z[zid] + s_items[8]);

    // Compute final potential energy
    V[gid] = 0.5 * s_items[9] * ((V_x * V_x + V_y * V_y + V_z * V_z)
                + (Ax[gid] * Ax[gid] + Ay[gid] * Ay[gid] + Az[gid] * Az[gid]));
}