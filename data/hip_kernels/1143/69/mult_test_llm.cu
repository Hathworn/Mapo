#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double2 mult(double2 a, double2 b){
    return {a.x*b.x - a.y*b.y, a.x*b.y + a.y*b.x};
}

__device__ double2 mult(double2 a, double b){
    return {a.x*b, a.y*b};
}

__global__ void mult_test(double2 *a, double b, double2 *c){
    // Using threadIdx.x to compute in parallel for better performance
    int idx = threadIdx.x;
    c[idx] = mult(a[idx], b);
}