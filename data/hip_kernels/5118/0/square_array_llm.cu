#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square_array(float *a, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll the loop to increase performance by reducing branch divergence
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < N; i += stride) {
        a[i] = a[i] * a[i];
    }
}