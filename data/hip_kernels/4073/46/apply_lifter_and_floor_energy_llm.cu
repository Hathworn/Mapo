#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void apply_lifter_and_floor_energy( int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;
    float *feats = features + frame * ldf;

    // Apply lifter coefficients using shared memory and loop unrolling
    extern __shared__ float shared_lifter_coeffs[];
    if (thread_id < num_cols) {
        shared_lifter_coeffs[thread_id] = lifter_coeffs[thread_id];
    }
    __syncthreads();
    
    if (cepstral_lifter != 0.0f) {
        #pragma unroll
        for (int c = thread_id; c < num_cols; c += blockDim.x) {
            float lift = shared_lifter_coeffs[c];
            feats[c] *= lift;
        }
    }

    // Use first thread in each block to apply energy safely
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = logf(energy_floor);
        feats[0] = (energy_floor > 0.0f && energy < log_energy_floor) ? log_energy_floor : energy;
    }
}