#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    int total_threads = gridDim.x * blockDim.x;
    const float *Ar = A_in + blockIdx.x * ldi;
    float *Aw = A_out + blockIdx.x * ldo;

    int half_length = row_length / 2;
    for (int idx = thread_id; idx < half_length; idx += total_threads) {
        if (idx == 0) continue;  // ignore special case

        // Load complex value and calculate magnitude
        float2 val = reinterpret_cast<const float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        
        // Store calculated power or magnitude
        Aw[idx] = use_power ? ret : sqrtf(ret);
    }

    // Handle special case only once per block
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        // Store the power or magnitude of the DC and Nyquist components
        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}