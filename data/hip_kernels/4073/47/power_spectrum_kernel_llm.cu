#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    // Use shared memory for faster access
    __shared__ float2 shared_Ar[CU1DBLOCK];
    
    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {
        // Load data into shared memory
        if (idx < blockDim.x) {
            shared_Ar[thread_id] = reinterpret_cast<float2 *>(Ar)[idx];
        }
        __syncthreads();

        if (idx == 0) continue;

        // Use shared memory instead of global memory
        float2 val = shared_Ar[thread_id];
        float ret = val.x * val.x + val.y * val.y;
        if (use_power) {
            Aw[idx] = ret;
        } else {
            Aw[idx] = sqrtf(ret);
        }
    }

    // Handle special cases
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        if (use_power) {
            Aw[0] = real * real;
            Aw[half_length] = im * im;
        } else {
            Aw[0] = fabs(real);
            Aw[half_length] = fabs(im);
        }
    }
}
```
