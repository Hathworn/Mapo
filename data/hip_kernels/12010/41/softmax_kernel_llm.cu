#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    extern __shared__ float shared_input[];
    int tid = threadIdx.x;
    float sum = 0;
    float largest = -INFINITY;

    // Load data to shared memory
    if (tid < n) {
        shared_input[tid] = input[tid];
    }
    __syncthreads();

    // Find largest element
    for (int i = tid; i < n; i += blockDim.x) {
        float val = shared_input[i];
        largest = (val > largest) ? val : largest;
    }

    // Synchronize threads before proceeding
    __syncthreads();

    // Calculate exponentials and sum
    for (int i = tid; i < n; i += blockDim.x) {
        float e = exp((shared_input[i] - largest) / temp);
        sum += e;
        shared_input[i] = e;  // update shared memory
    }
    __syncthreads();

    // Normalize the output
    for (int i = tid; i < n; i += blockDim.x) {
        output[i] = shared_input[i] / sum;
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output)
{
    int b = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (b >= batch) return;

    // Execute softmax_device with shared memory
    extern __shared__ float shared_input[];
    softmax_device(n, input + b * offset, temp, output + b * offset);
}
```
