#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Exit early if thread offset is outside of valid range
    if (offset >= size) return;

    // Compute index and update output with bias
    int index = (batch * n + filter) * size + offset;
    output[index] += biases[filter];
}