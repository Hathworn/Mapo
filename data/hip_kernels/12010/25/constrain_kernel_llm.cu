#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index
    int global_index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if(global_index < N) {
        int idx = global_index * INCX;
        
        // Use fmaxf function for boundary constraints
        X[idx] = fmaxf(-ALPHA, fminf(ALPHA, X[idx]));
    }
}