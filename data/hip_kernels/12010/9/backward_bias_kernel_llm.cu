#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    
    float sum = 0.0f;
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) { // Use thread stride loop for better memory access
            int index = i + size * (filter + n * b);
            sum += delta[index];
        }
    }
    part[p] = sum;

    __syncthreads();

    // Reduce within block using binary tree reduction
    for (int offset = BLOCK / 2; offset > 0; offset >>= 1) {
        if (p < offset) {
            part[p] += part[p + offset];
        }
        __syncthreads();
    }

    if (p == 0) {
        atomicAdd(&bias_updates[filter], part[0]); // Use atomicAdd for concurrent writes
    }
}