#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    
    float sum = 0.0f; // Use register for accumulating sum

    // Loop unrolling for spatial
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) { // Stride by number of threads
            int index = j * spatial * filters + filter * spatial + i;
            sum += pow((x[index] - mean[filter]), 2);
        }
    }
    
    local[id] = sum;
    __syncthreads();
    
    // Reduction within block
    if (id == 0) {
        sum = 0.0f;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        variance[filter] = sum / (spatial * batch - 1);
    }
}