#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    int stride = blockDim.x * gridDim.x;           // Calculate stride for grid stride loop
    for (; i < N; i += stride) {                   // Use grid stride looping for better parallelism
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];   // Copy elements from X to Y
    }
}