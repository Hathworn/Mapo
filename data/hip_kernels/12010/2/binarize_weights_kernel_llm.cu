#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Use shared memory for intermediate summation results to improve memory access efficiency
    extern __shared__ float shared_mean[];
    
    // Each thread computes a partial sum
    float local_mean = 0.0f;
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        local_mean += abs(weights[f * size + i]);
    }
    
    // Atomic add to accumulate the results of each thread in the block
    atomicAdd(&shared_mean[threadIdx.x], local_mean);

    __syncthreads();

    // Single thread computes final mean for each block
    if (threadIdx.x == 0) {
        float block_mean = 0.0f;
        for (int i = 0; i < blockDim.x; ++i) {
            block_mean += shared_mean[i];
        }
        block_mean /= size;
        shared_mean[0] = block_mean;
    }

    __syncthreads();

    // All threads use the mean value computed above
    float mean = shared_mean[0];
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    }
}