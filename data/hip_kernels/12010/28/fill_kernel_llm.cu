#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride to handle blocks larger than N
    for (int i = idx; i < N; i += gridDim.x * blockDim.x) {
        X[i * INCX] = ALPHA;
    }
}