#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride approach for thread block to cover all n elements
    for (; i < n; i += blockDim.x * gridDim.x) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);

        if(abs_val < 1.0f) {
            error[i] = diff * diff;
            delta[i] = diff;
        }
        else {
            error[i] = 2.0f * abs_val - 1.0f;
            delta[i] = (diff < 0.0f) ? -1.0f : 1.0f;
        }
    }
}