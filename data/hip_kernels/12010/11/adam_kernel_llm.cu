#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    // Use a single variable for frequent calculations
    float B1t = pow(B1, t);
    float B2t = pow(B2, t);
    float rate_adj = rate * sqrt(1.f - B2t) / (1.f - B1t);

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) { // Change to '<' for better loop structure
        float sqrt_v = sqrt(v[index]) + eps;
        x[index] -= rate_adj * m[index] / sqrt_v;
    }
}