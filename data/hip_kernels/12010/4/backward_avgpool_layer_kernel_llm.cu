#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n * c) return;  // Unroll grid dimension combination
    
    int k = id % c;
    int b = id / c;

    int out_index = k + c * b;
    int num_elements = w * h;
    float out_val = out_delta[out_index] / num_elements; // Precompute division

    for (int i = 0; i < num_elements; ++i) {
        int in_index = i + num_elements * (k + b * c);
        atomicAdd(&in_delta[in_index], out_val); // Atomic add to handle race conditions
    }
}