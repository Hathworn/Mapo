#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK 256

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates) 
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0.0f;

    // Unroll loop and combine index calculation to optimize memory access
    for (int b = 0; b < batch; b++) {
        for (int i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index] * x_norm[index];
        }
    }
    part[p] = sum;

    __syncthreads();

    // Use reduction to accumulate results for better performance
    if (p < 128) part[p] += part[p + 128];
    __syncthreads();
    if (p < 64) part[p] += part[p + 64];
    __syncthreads();
    if (p < 32) part[p] += part[p + 32];
    __syncthreads();
    if (p < 16) part[p] += part[p + 16];
    __syncthreads();
    if (p < 8) part[p] += part[p + 8];
    __syncthreads();
    if (p < 4) part[p] += part[p + 4];
    __syncthreads();
    if (p < 2) part[p] += part[p + 2];
    __syncthreads();
    if (p < 1) part[p] += part[p + 1];
    __syncthreads();

    // Write the accumulated result to the global memory
    if (p == 0) {
        scale_updates[filter] += part[0];
    }
}