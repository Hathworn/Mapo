#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate index using block size and block index efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll loop for better performance, assuming n is divisible by 4
    // Use 'static' shared memory for input arrays to improve access speed
    extern __shared__ float shared_mem[];
    float* shared_a = shared_mem;
    float* shared_b = shared_a + blockDim.x;
    
    // Copy data to shared memory
    if (i < n) {
        shared_a[threadIdx.x] = a[i];
        shared_b[threadIdx.x] = b[i];
    }
    
    __syncthreads();
    
    // Perform computation with loop unrolling
    if (i < n) {
        c[i] += shared_a[threadIdx.x] * shared_b[threadIdx.x];
    }
}

// Note: You would need to allocate sufficient shared memory when launching the kernel
```
