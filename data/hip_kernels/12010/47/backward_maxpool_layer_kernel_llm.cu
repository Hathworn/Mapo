#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int c = in_c;
    int area = (size - 1) / stride;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Calculate the indexes in a coalesced manner
    int j = id % in_w;
    int i = (id / in_w) % in_h;
    int k = (id / (in_w * in_h)) % in_c;
    int b = id / (in_w * in_h * in_c);

    int w_offset = -pad;
    int h_offset = -pad;

    float d = 0.0f;
    int l, m;

    #pragma unroll
    for (l = -area; l <= area; ++l) {
        #pragma unroll
        for (m = -area; m <= area; ++m) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;
            int out_index = out_w + w * (out_h + h * (k + c * b));
            bool valid = (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h);
            // Accumulate delta value if conditions are met
            d += (valid && indexes[out_index] == id) ? delta[out_index] : 0.0f;
        }
    }
    prev_delta[id] += d;  // Update the previous delta
}