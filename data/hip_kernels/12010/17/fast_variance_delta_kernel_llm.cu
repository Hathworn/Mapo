#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta) {
    extern __shared__ float local[]; // Use dynamic shared memory

    int id = threadIdx.x;
    int filter = blockIdx.x;

    // Initialize local memory
    local[id] = 0;

    // Iterate over the batch and spatial dimensions
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += blockDim.x) { // Improve loop stride
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += delta[index] * (x[index] - mean[filter]);
        }
    }

    // Reduce within the block
    __syncthreads(); // Ensure all threads have completed their additions
    if (id == 0) {
        float reduceSum = 0;
        for (int i = 0; i < blockDim.x; ++i) {
            reduceSum += local[i];
        }
        variance_delta[filter] = reduceSum * -.5f * powf(variance[filter] + .000001f, -1.5f);
    }
}