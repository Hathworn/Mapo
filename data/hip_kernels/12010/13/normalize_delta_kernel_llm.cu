#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use loop to cover all elements, especially useful when N is large
    for (; index < N; index += stride) {
        int f = (index/spatial)%filters;
        float sqrt_variance = sqrt(variance[f]) + .000001f;
        float common_factor = 1.f / (spatial * batch);
        float x_mean_diff = x[index] - mean[f];

        // Optimize computation by pre-calculating common terms
        delta[index] = delta[index] / sqrt_variance 
                       + variance_delta[f] * 2.f * x_mean_diff * common_factor 
                       + mean_delta[f] * common_factor;
    }
}