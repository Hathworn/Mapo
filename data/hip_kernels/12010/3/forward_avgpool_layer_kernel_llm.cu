#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    // Calculate the unique thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    // Determine the channel and batch index
    int k = id % c;
    id /= c;
    int b = id;

    // Compute the output index
    int out_index = (k + c * b);
    output[out_index] = 0.0f;

    // Perform accumulation using shared memory to reduce global memory operations
    extern __shared__ float shared_input[];
    int shared_idx = threadIdx.x;
    for (int i = shared_idx; i < w * h; i += blockDim.x) {
        int in_index = i + h * w * (k + b * c);
        shared_input[shared_idx] += input[in_index];
    }
    __syncthreads();

    // Sum up the shared memory results for final output computation
    if (shared_idx == 0) {
        for (int i = 0; i < blockDim.x; ++i) {
            output[out_index] += shared_input[i];
        }
        output[out_index] /= (w * h);
    }
}