#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1.f / (batch * spatial - 1);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float var_sum = 0.0f; // Declare a local variable for variance accumulation

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            float diff = (x[index] - mean[i]);
            var_sum += diff * diff; // Accumulate squared differences
        }
    }
    variance[i] = var_sum * scale; // Apply scaling to the local accumulation
}