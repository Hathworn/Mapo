#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    float h = hsv.x, s = hsv.y, v = hsv.z;
    float r, g, b;

    if (s == 0) {
        r = g = b = v;
    } else {
        int index = (int)floorf(h);
        float f = h - index;
        float p = v * (1 - s);
        float q = v * (1 - s * f);
        float t = v * (1 - s * (1 - f));

        if (index == 0)      { r = v; g = t; b = p; }
        else if (index == 1) { r = q; g = v; b = p; }
        else if (index == 2) { r = p; g = v; b = t; }
        else if (index == 3) { r = p; g = q; b = v; }
        else if (index == 4) { r = t; g = p; b = v; }
        else                 { r = v; g = p; b = q; }
    }

    // Clamp RGB values
    r = fminf(fmaxf(r, 0.0f), 1.0f);
    g = fminf(fmaxf(g, 0.0f), 1.0f);
    b = fminf(fmaxf(b, 0.0f), 1.0f);

    return make_float3(r, g, b);
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    float r = rgb.x, g = rgb.y, b = rgb.z;
    float h, s, v;
    float max = fmaxf(r, fmaxf(g, b));
    float min = fminf(r, fminf(g, b));
    float delta = max - min;
    v = max;

    if (max == 0) {
        s = 0;
        h = -1;
    } else {
        s = delta / max;

        if (r == max) {
            h = (g - b) / delta;
        } else if (g == max) {
            h = 2 + (b - r) / delta;
        } else {
            h = 4 + (r - g) / delta;
        }

        if (h < 0) h += 6;
    }

    return make_float3(h, s, v);
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int size = batch * w * h;
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    if (id >= size) return;

    int x = id % w;
    int y = (id / w) % h;
    int batch_offset = id / (w * h);

    float rshift = rand[0];
    float gshift = rand[1];
    float bshift = rand[2];

    // Simplified random access
    float r0 = rand[8 * batch_offset + 0];
    float r1 = rand[8 * batch_offset + 1];
    float r2 = rand[8 * batch_offset + 2];
    float r3 = rand[8 * batch_offset + 3];

    saturation = (r1 > 0.5f) ? 1.0f / (r0 * (saturation - 1) + 1) : r0 * (saturation - 1) + 1;
    exposure = (r3 > 0.5f) ? 1.0f / (r2 * (exposure - 1) + 1) : r2 * (exposure - 1) + 1;

    size_t offset = batch_offset * h * w * 3;
    image += offset;

    float r = image[x + w * (y + h * 0)];
    float g = image[x + w * (y + h * 1)];
    float b = image[x + w * (y + h * 2)];
    float3 rgb = make_float3(r, g, b);

    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }

    image[x + w * (y + h * 0)] = rgb.x * scale + translate + (rshift - 0.5f) * shift;
    image[x + w * (y + h * 1)] = rgb.y * scale + translate + (gshift - 0.5f) * shift;
    image[x + w * (y + h * 2)] = rgb.z * scale + translate + (bshift - 0.5f) * shift;
}