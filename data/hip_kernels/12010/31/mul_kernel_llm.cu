#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Utilize more threads per block by using a 2D grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int index = idy * gridDim.x * blockDim.x + idx;
    
    if(index < N) {
        Y[index * INCY] *= X[index * INCX];
    }
}