#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop to handle more data with fewer threads
    for (; i < n; i += blockDim.x * gridDim.x) {
        c[i] = s[i] * a[i] + (1 - s[i]) * (b ? b[i] : 0);
    }
}