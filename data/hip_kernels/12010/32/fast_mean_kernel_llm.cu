#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0.0f;

    int filter = blockIdx.x;
 
    int j;
    for(j = 0; j < batch; ++j){
        for(int i = id; i < spatial; i += threads){
            // Combine the loop increment and index calculation
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += x[index];
        }
    }

    __syncthreads();  // Synchronize threads before reduction

    // Reduction with parallel summation
    for(int stride = threads / 2; stride > 0; stride >>= 1) {
        if(id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();  // Ensure all additions are done before next stride
    }

    if(id == 0) {
        mean[filter] = local[0] / (spatial * batch);  // Directly compute the mean
    }
}