#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;

    // Use shared memory to reduce global memory accesses
    extern __shared__ float shared_data[];
    
    float sum = 0;
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        sum += abs(input[i * size + s]);
    }
    shared_data[threadIdx.x] = sum;
    __syncthreads();

    // Reduction in shared memory to calculate mean
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            shared_data[threadIdx.x] += shared_data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    float mean = shared_data[0] / n;
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}