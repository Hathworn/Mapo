#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Use stride loop to cover all elements
    int stride = blockDim.x * gridDim.x;
    for (int idx = i; idx < N; idx += stride) {
        Y[OFFY + idx * INCY] += ALPHA * X[OFFX + idx * INCX];
    }
}