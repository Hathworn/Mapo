#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Compute the global index for this thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we only access valid indices
    if(i < N) {
        // Use shared memory to reduce repeated global memory accesses
        // Cache the value of X[i*INCX] for repeated use, improving performance
        float x_val = X[i*INCX];
        
        // Compare cached value to ALPHA squared and update X using conditional
        if((x_val * x_val) < (ALPHA * ALPHA)) {
            X[i*INCX] = 0;
        }
    }
}