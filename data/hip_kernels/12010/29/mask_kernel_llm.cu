#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    // Calculate the global index using a more efficient method
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    if (i < n && mask[i] == mask_num)
    {
        // Write the masked number to the output if conditions are met
        x[i] = mask_num;
    }
}