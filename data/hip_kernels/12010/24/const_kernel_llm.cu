#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Use a strided loop for more efficient memory access
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x * gridDim.y)
    {
        X[idx * INCX] = ALPHA;  // Assign ALPHA to element at the updated index
    }
}