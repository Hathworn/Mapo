#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified calculation of global index
    if (index < N) {
        int f = (index / spatial) % filters; // Calculate filter index
        float m = mean[f]; // Load mean into register
        float v = variance[f]; // Load variance into register

        // Normalize x[index]
        x[index] = (x[index] - m) / (sqrt(v) + 0.000001f);
    }
}