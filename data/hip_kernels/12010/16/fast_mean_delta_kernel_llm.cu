#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Use grid-stride loop for better memory access pattern
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j*spatial*filters + filter*spatial + i;
            local[id] += delta[index];
        }
    }

    // Parallel reduction to accumulate results
    __syncthreads();
    for (int offset = threads / 2; offset > 0; offset >>= 1) {
        if (id < offset) {
            local[id] += local[id + offset];
        }
        __syncthreads();
    }

    // Normalize by variance and update mean_delta
    if (id == 0) {
        mean_delta[filter] = local[0] * (-1.0f / sqrtf(variance[filter] + 0.000001f));
    }
}