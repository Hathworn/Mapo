#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;
    
    // Use shared memory for faster access
    extern __shared__ float sharedSum[];
    sharedSum[threadIdx.x] = 0.0f;
    
    // Unroll the loop for better performance
    for(int k = threadIdx.x; k < n; k += blockDim.x){
        sharedSum[threadIdx.x] += x[k*groups + i];
    }
    
    // Use parallel reduction to sum up results
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sharedSum[threadIdx.x] += sharedSum[threadIdx.x + s];
        }
        __syncthreads();
    }
    
    // Write result to global memory by the first thread in the block
    if (threadIdx.x == 0) {
        sum[i] = sharedSum[0];
    }
}