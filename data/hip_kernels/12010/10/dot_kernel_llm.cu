#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate f1 and f2 from the index
    int num_pairs = n * (n - 1) / 2;
    if (index >= num_pairs) return;  // Early exit if index out of bounds
    
    int f1, f2;
    for (f1 = 0, f2 = 1; f1 < n - 1; ++f1) {
        if (index < (n - f1 - 1)) {
            f2 = f1 + index + 1;
            break;
        }
        index -= (n - f1 - 1);
    }

    float sum = 0.0f;
    float norm1 = 0.0f;
    float norm2 = 0.0f;
    
    // Perform computation for each batch
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            // Compute indices based on current f1, f2, and batch
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            
            float val1 = output[i1];
            float val2 = output[i2];
            
            sum += val1 * val2;
            norm1 += val1 * val1;
            norm2 += val2 * val2;
        }
    }
    
    // Compute norms and normalize sum
    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    if (norm == 0.0f) return;  // Avoid division by zero

    sum /= norm;
    
    // Update delta values
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            
            delta[i1] += -scale * sum * output[i2] / norm;
            delta[i2] += -scale * sum * output[i1] / norm;
        }
    }
}