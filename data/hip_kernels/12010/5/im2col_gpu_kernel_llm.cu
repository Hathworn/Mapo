#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void im2col_gpu_kernel(const int n, const float* __restrict__ data_im, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float * __restrict__ data_col) {
    // Calculate unique thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll the loop to improve performance
    for (; index < n; index += blockDim.x * gridDim.x) {
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        float* data_col_ptr = data_col + (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im + (channel_in * height + h_in) * width + w_in;
        #pragma unroll
        for (int i = 0; i < ksize; ++i) {
            #pragma unroll
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;
                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ? data_im_ptr[i * width + j] : 0;
                data_col_ptr += height_col * width_col;
            }
        }
    }
}