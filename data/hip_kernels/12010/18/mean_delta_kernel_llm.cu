#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Using shared memory for mean_delta to optimize memory access
    extern __shared__ float shared_mean_delta[];
    shared_mean_delta[threadIdx.x] = 0;

    // Unroll loop for faster accumulation
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            shared_mean_delta[threadIdx.x] += delta[index];
        }
    }

    // Multiply outside the loop for reduced operations
    shared_mean_delta[threadIdx.x] *= (-1.f / sqrtf(variance[i] + 0.000001f));

    // Write result back to global memory
    mean_delta[i] = shared_mean_delta[threadIdx.x];
}