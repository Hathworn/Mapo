#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    int stride = blockDim.x * gridDim.x; // Calculate stride for loop

    for (; i < n; i += stride) { // Use stride loop for better utilization
        binary[i] = (x[i] >= 0) ? 1 : -1;
    }
}