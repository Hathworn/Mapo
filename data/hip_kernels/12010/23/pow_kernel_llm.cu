#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;
    
    // Loop through elements with step size of gridSize
    for(; i < N; i += gridSize)
    {
        Y[i * INCY] = powf(X[i * INCX], ALPHA); // Use powf for single-precision
    }
}