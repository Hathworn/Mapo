#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.0f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (i >= filters) return;
    
    float sum = 0.0f; // Use register for sum to reduce global memory accesses

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += x[index]; // Accumulate sum
        }
    }

    mean[i] = sum * scale; // Update mean with scaled sum
}