#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Compute the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    // Reduce frequency of the expensive integer division and modulus operations
    int in_index = idx;
    int in_w = in_index % w;
    int in_h = (in_index / w) % h;
    int in_c = (in_index / (w * h)) % c;
    int b = (in_index / (w * h * c)) % batch;

    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;

    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    // Use ternary operator for branchless conditional assignment
    out[forward ? out_index : in_index] = x[forward ? in_index : out_index];
}