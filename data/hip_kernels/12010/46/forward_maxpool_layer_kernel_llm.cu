#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    // Calculate output dimensions
    int h = (in_h + 2*pad) / stride;
    int w = (in_w + 2*pad) / stride;
    
    // Calculate unique global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Derive indices for batch, channel, height, and width
    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    // Calculate padding offsets
    int w_offset = -pad;
    int h_offset = -pad;

    // Initialize output index and max values
    int out_index = j + w * (i + h * (k + in_c * b));
    float max_val = -INFINITY;
    int max_index = -1;

    // Loop unrolling to improve performance
    #pragma unroll
    for (int l = 0; l < size; ++l) {
        #pragma unroll
        for (int m = 0; m < size; ++m) {
            int cur_h = h_offset + i * stride + l;
            int cur_w = w_offset + j * stride + m;
            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            int valid = (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w);
            float val = valid ? input[index] : -INFINITY;

            // Update max value and index
            if (val > max_val) {
                max_val = val;
                max_index = index;
            }
        }
    }

    // Assign output values
    output[out_index] = max_val;
    indexes[out_index] = max_index;
}