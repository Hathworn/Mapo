#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float var_inv3 = -0.5 * powf(variance[i] + 0.000001f, -1.5f);  // Precompute inverse of variance raised to power
    
    float accum = 0.0f;  // Accumulator
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            accum += delta[index] * (x[index] - mean[i]);  // Accumulate delta
        }
    }
    variance_delta[i] = accum * var_inv3;
}