#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop with stride to handle more elements per thread
    for(int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        X[idx * INCX] *= ALPHA;
    }
}