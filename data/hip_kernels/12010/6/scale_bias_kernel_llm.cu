#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    // Calculate global thread index and filter value
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;

    // Only proceed if idx is within valid range
    if(idx < size) {
        // Calculate output index and apply bias scaling
        int output_idx = (blockIdx.z * n + filter) * size + idx;
        output[output_idx] *= biases[filter];
    }
}