#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate unique global thread index in the grid
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Process only valid indices
    if (i < n) {
        float delta_c = dc[i];
        float s_val = s[i];

        if (da) {
            da[i] += delta_c * s_val;     // Compute da only if da is not null
        }
        db[i] += delta_c * (1 - s_val);  // Update db
        ds[i] += delta_c * (a[i] - b[i]); // Update ds with calculated value
    }
}