#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread ID to avoid repeated division in a parallel environment
    int idx = threadIdx.x;
    if (idx == 0) {
        *accuracy /= N;
    }
}