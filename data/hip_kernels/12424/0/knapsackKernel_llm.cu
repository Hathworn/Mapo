#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define index(i, j, N)  ((i)*(N+1)) + (j)

__device__ int maximum(int a, int b) {
    return (a > b)? a : b;
}

__global__ void knapsackKernel(int *profits, int *weights, int *f, int capacity, int i) {
    int c = threadIdx.x;
    int idx = index(i, c, capacity); // Pre-compute index for current item

    if (i == 0 || c == 0) {
        f[idx] = 0; // Initialize base case
    } else {
        int idx_prev = index(i - 1, c, capacity); // Pre-compute previous row index
        if (weights[i - 1] <= c) {
            // Utilize pre-computed index for better readability
            f[idx] = maximum(f[idx_prev], profits[i - 1] + f[index(i - 1, c - weights[i - 1], capacity)]);
        } else {
            f[idx] = f[idx_prev];
        }
    }
}