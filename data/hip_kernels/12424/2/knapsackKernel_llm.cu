#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void knapsackKernel(int *profits, int *weights, int *input_f, int *output_f, int capacity, int c_min, int k) {
    // Calculate global thread index
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit early if the index is out of bounds
    if (c < c_min || c > capacity) return;

    // Read values only once for efficiency
    int weight_k = weights[k - 1];
    int profit_k = profits[k - 1];
    int input_val = input_f[c];

    // Perform computation and update output
    if (c >= weight_k && input_val < input_f[c - weight_k] + profit_k) {
        output_f[c] = input_f[c - weight_k] + profit_k;
    } else {
        output_f[c] = input_val;
    }
}