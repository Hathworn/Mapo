#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmUpGPU()
{
    // The kernel is used for GPU warm-up and deliberately does nothing,
    // there is no need for optimization as the function is intentionally empty.
}