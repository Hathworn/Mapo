#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void knapsackKernel(int *profits, int *weights, int *input_f, int *output_f, int capacity, int c_min, int k) {
    // Calculate global index
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if the current index is within bounds
    if (c >= c_min && c <= capacity) {
        // Calculate potential new value
        int new_value = (c >= weights[k-1]) ? input_f[c - weights[k-1]] + profits[k-1] : input_f[c];
        
        // Write the maximum value to output
        output_f[c] = max(input_f[c], new_value);
    }
}