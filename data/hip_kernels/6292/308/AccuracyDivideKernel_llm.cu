#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Only perform division once by a single thread to prevent race conditions
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}