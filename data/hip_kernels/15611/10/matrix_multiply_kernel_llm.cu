#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    // Cache input vector in shared memory for faster access
    extern __shared__ double shared_vector[];
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    if (threadIdx.x < dim_mn) {
        shared_vector[threadIdx.x] = vector_in[threadIdx.x];
    }
    __syncthreads(); // Ensure shared memory is fully populated
    
    if (i < dim_mn) {
        double out = 0.0;
        // Access shared memory instead of global memory for vector_in
        for (long j = 0; j < dim_mn; j++) {
            out += matrix[i * dim_mn + j] * shared_vector[j];
        }
        vector_out[i] = out;
    }
}