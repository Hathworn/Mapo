#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bigstencil(int* in, int* out, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index is within bounds
    if (i < n) {
        out[i] = in[i] + 2;
    }
}