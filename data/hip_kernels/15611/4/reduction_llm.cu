#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(int* input, int* output) {
    __shared__ int tmp[TPB];

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    tmp[threadIdx.x] = input[idx];

    __syncthreads();

    // Perform reduction in a loop to handle arbitrary block sizes
    for(int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if(threadIdx.x < stride) {
            tmp[threadIdx.x] += tmp[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if(threadIdx.x == 0) {
        output[blockIdx.x] = tmp[0];
    }
}