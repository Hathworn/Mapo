#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Use the shared memory to reduce global memory accesses
    extern __shared__ short2 shared_loc[];

    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Load data into shared memory to coalesce global memory access
        shared_loc[threadIdx.x] = loc_[ptidx];
        __syncthreads();

        short2 loc = shared_loc[threadIdx.x];

        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}