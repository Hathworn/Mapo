#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < h && j < w) { // Combined boundary check for better readability
        int pos = i * w + j; // Use int instead of const to avoid unnecessary storage optimization
        image[pos] = value; // Assign value if within bounds
    }
}