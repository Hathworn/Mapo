#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use of stride loop to handle larger vectors efficiently
    for (int i = pos; i < len; i += blockDim.x * gridDim.x) 
    {
        d_res[i] = d_src[i] * scale;
    }
}