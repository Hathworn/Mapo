#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global thread position optimized using built-in variables
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure threads within image bounds
    if (i < h && j < w) {
        // Flattened array index
        const int pos = i * s + j;

        // Load normalization factor once
        float scale = normalization_factor[pos];

        // Calculate inverse scale and handle zero-scale case
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

        // Apply normalization
        image[pos] *= invScale;
    }
}