#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // pixel containing bottom left corner
    float px, py;
    float dx = modff (cx, &px);
    float dy = modff (cy, &py);

    // target pixel integer coords
    int tx = (int) px;
    int ty = (int) py;
    float value = src[image_row_offset + j];
    float weight;

    // combined if statements for pixel contribution calculation
    bool inBoundsBL = !(tx >= w || tx < 0 || ty >= h || ty < 0);
    bool inBoundsBR = !(tx - 1 >= w || tx - 1 < 0 || ty >= h || ty < 0);
    bool inBoundsUL = !(tx - 1 >= w || tx - 1 < 0 || ty - 1 >= h || ty - 1 < 0);
    bool inBoundsUR = !(tx >= w || tx < 0 || ty - 1 >= h || ty - 1 < 0);

    if (inBoundsBL)
    {
        weight = dx * dy;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    if (inBoundsBR)
    {
        weight = (1.0f - dx) * dy;
        _atomicAdd(dst + ty * image_stride + (tx - 1), value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + (tx - 1), weight);
    }

    if (inBoundsUL)
    {
        weight = (1.0f - dx) * (1.0f - dy);
        _atomicAdd(dst + (ty - 1) * image_stride + (tx - 1), value * weight);
        _atomicAdd(normalization_factor + (ty - 1) * image_stride + (tx - 1), weight);
    }

    if (inBoundsUR)
    {
        weight = dx * (1.0f - dy);
        _atomicAdd(dst + (ty - 1) * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + (ty - 1) * image_stride + tx, weight);
    }
}