#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ResetHeap_kernel(int *mplHeap, int *mplHeapPtr, int numBlock)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= numBlock)
        return;

    // Initialize heap pointer only once outside the loop
    if (index == 0) 
    {
        mplHeapPtr[0] = numBlock - 1;
    }

    // Calculate heap index more efficiently
    mplHeap[index] = numBlock - index - 1;
}