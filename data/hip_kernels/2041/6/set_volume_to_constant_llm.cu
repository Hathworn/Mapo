#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_volume_to_constant(float * out, float value)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Use early exit to reduce nesting and improve readability
    if (i < c_Size.x && j < c_Size.y && k < c_Size.z) 
    {
        long int id = (k * c_Size.y + j) * c_Size.x + i;
        out[id] = value;
    }
}