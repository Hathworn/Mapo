#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtract_kernel(float * in1, float * in2, float * out)
{
    // Optimize index calculation to avoid repeated multiplications and improve readability.
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Use shared memory to limit global memory access when possible.
    __shared__ float shared_in1[BLOCK_SIZE];
    __shared__ float shared_in2[BLOCK_SIZE];

    if (i < c_Size.x && j < c_Size.y && k < c_Size.z)
    {
        long int id = (k * c_Size.y + j) * c_Size.x + i;

        shared_in1[threadIdx.x] = in1[id];
        shared_in2[threadIdx.x] = in2[id];

        out[id] = shared_in1[threadIdx.x] - shared_in2[threadIdx.x];
    }
}