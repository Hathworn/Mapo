#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiply_by_beta_kernel(float *input, float *output, float beta, dim3 size) {
    // Calculate global thread index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Check if the thread is within bounds
    if (i < size.x && j < size.y && k < size.z) {
        // Calculate linear index and multiply by beta
        long int id = (k * size.y + j) * size.x + i;
        output[id] = input[id] * beta;
    }
}