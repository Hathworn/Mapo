#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gradient_and_subtract_kernel(float *in, float *grad_x, float *grad_y, float *grad_z)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z) return;

    long int id = (k * c_Size.y + j) * c_Size.x + i;

    // Avoid redundant memory access by using shared memory for neighboring elements
    __shared__ float s_in_x[BLOCK_SIZE_X + 1];
    __shared__ float s_in_y[BLOCK_SIZE_Y + 1];
    __shared__ float s_in_z[BLOCK_SIZE_Z + 1];

    if (i < (c_Size.x - 1))
        s_in_x[threadIdx.x] = in[id + 1];
    if (j < (c_Size.y - 1))
        s_in_y[threadIdx.y] = in[id + c_Size.x];
    if (k < (c_Size.z - 1))
        s_in_z[threadIdx.z] = in[id + c_Size.x * c_Size.y];

    __syncthreads();

    if (i < (c_Size.x - 1))
        grad_x[id] -= ((s_in_x[threadIdx.x] - in[id]) / c_Spacing.x);
    if (j < (c_Size.y - 1))
        grad_y[id] -= ((s_in_y[threadIdx.y] - in[id]) / c_Spacing.y);
    if (k < (c_Size.z - 1))
        grad_z[id] -= ((s_in_z[threadIdx.z] - in[id]) / c_Spacing.z);
}