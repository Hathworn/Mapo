#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_normalize_and_add_to_output(float * __restrict__ dev_vol_in, float * __restrict__ dev_vol_out, float * __restrict__ dev_accumulate_weights, float * __restrict__ dev_accumulate_values)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index computation
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < c_volSize.x && j < c_volSize.y && k < c_volSize.z) // Combined condition check
    {
        long int out_idx = i + (j + k * c_volSize.y) * c_volSize.x; // Row-major indexing

        float eps = 1e-6;

        float weight = dev_accumulate_weights[out_idx];
        float value_to_add = (abs(weight) > eps) ? (dev_accumulate_values[out_idx] / weight) : 0.0f; // Compute value to add conditionally

        dev_vol_out[out_idx] = dev_vol_in[out_idx] + (c_normalize ? value_to_add : dev_accumulate_values[out_idx]); // Simplified conditional addition
    }
}