#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void magnitude_threshold_kernel(float * grad_x, float * grad_y, float * grad_z, float gamma)
{
    // Use shared memory for optimization
    __shared__ float s_grad_x[BLOCK_SIZE];
    __shared__ float s_grad_y[BLOCK_SIZE];
    __shared__ float s_grad_z[BLOCK_SIZE];

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;
    
    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
        return;

    long int id = (k * c_Size.y + j) * c_Size.x + i;

    // Load data into shared memory
    s_grad_x[threadIdx.x] = grad_x[id];
    s_grad_y[threadIdx.x] = grad_y[id];
    s_grad_z[threadIdx.x] = grad_z[id];

    __syncthreads();  // Ensure all threads have loaded their data

    float norm = sqrt(s_grad_x[threadIdx.x] * s_grad_x[threadIdx.x] +
                      s_grad_y[threadIdx.x] * s_grad_y[threadIdx.x] +
                      s_grad_z[threadIdx.x] * s_grad_z[threadIdx.x]);

    if (norm > gamma)
    {
        float ratio = gamma / norm;
        s_grad_x[threadIdx.x] *= ratio;
        s_grad_y[threadIdx.x] *= ratio;
        s_grad_z[threadIdx.x] *= ratio;
    }

    // Write modified data back to global memory
    grad_x[id] = s_grad_x[threadIdx.x];
    grad_y[id] = s_grad_y[threadIdx.x];
    grad_z[id] = s_grad_z[threadIdx.x];
}