#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    // Use shared memory to reduce global memory access
    __shared__ unsigned int sharedMinmax[32 * 16];

    const int tx = threadIdx.x;
    int numPts = 0;
    
    for (int y = 0; y < h / 32; y++) {
        // Load minmax values into shared memory
        for (int x = 0; x < w; x += 16) {
            sharedMinmax[tx] = minmax[y * w + x + tx];
            __syncthreads();  // Synchronize threads to ensure shared memory is updated

            unsigned int val = sharedMinmax[tx];

            if (val) {
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        // Calculate and store index using shared memory
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
            __syncthreads();  // Synchronize threads after processing
        }
    }
}