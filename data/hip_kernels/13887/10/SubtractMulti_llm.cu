#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Calculate linear index directly for better memory coalescing
    const int x = blockIdx.x * SUBTRACTM_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACTM_H + threadIdx.y;

    if (x < width && y < height)
    {
        int sz = height * pitch;
        int p = blockIdx.z * sz + y * pitch + x;
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
    // Removed __syncthreads() since no shared memory is being used
}