#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];

    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx; // Simplified index calculation
    const int y = blockIdx.y * 16;
    const int b = blockDim.x;
    int p = y * width + x;

    // Initialize shared memory
    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        float val = d_Data[p - x];
        minvals[tx] = val;
        maxvals[tx] = val;
    }

    // Unroll loop to improve performance
    #pragma unroll
    for (int ty = 1; ty < 16; ty++) {
        p += width;
        if (x < width) { // Fix index check to use x instead of tx
            float val = d_Data[p];
            if (val < minvals[tx]) minvals[tx] = val;
            if (val > maxvals[tx]) maxvals[tx] = val;
        }
    }

    __syncthreads();

    // Parallel reduction to find min and max
    for (int d = b >> 1; d > 0; d >>= 1) {
        if (tx < d) {
            if (minvals[tx + d] < minvals[tx]) minvals[tx] = minvals[tx + d];
            if (maxvals[tx + d] > maxvals[tx]) maxvals[tx] = maxvals[tx + d];
        }
        __syncthreads();
    }

    // Output results
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr + 0] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}