#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Calculate position using block and thread indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;
    
    // Check bounds and perform computation
    if (x < width && y < height) {
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
}