#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];

    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    int p = y * pitch + x;

    // Load data into shared memory, handle out-of-bounds case
    float val = (x < width) ? d_Data[p] : 0.0f; // Initialize as 0 for consistency
    minvals[tx] = val; 
    maxvals[tx] = val;

    // Loop unrolling for faster convergence, reduce condition checks
    for (int ty = 1; ty < 16; ty += 2) {
        p += pitch;
        if (tx < width) {
            val = d_Data[p];
            minvals[tx] = min(minvals[tx], val);
            maxvals[tx] = max(maxvals[tx], val);

            // Unrolled second iteration
            if (ty + 1 < 16) {
                p += pitch;
                val = d_Data[p];
                minvals[tx] = min(minvals[tx], val);
                maxvals[tx] = max(maxvals[tx], val);
            }
        }
    }
    __syncthreads();

    // Use efficient reduction within block
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (tx < offset) {
            minvals[tx] = min(minvals[tx], minvals[tx + offset]);
            maxvals[tx] = max(maxvals[tx], maxvals[tx + offset]);
        }
        __syncthreads();
    }

    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr + 0] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}