#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int pitch, int height) {
    // Data cache with reduced shared memory usage
    __shared__ float data1[3 * (MINMAX_W + 2)];
    __shared__ float data2[3 * (MINMAX_W + 2)];
    __shared__ float data3[3 * (MINMAX_W + 2)];
    __shared__ float ymin[(MINMAX_W + 2)];
    __shared__ float ymax[(MINMAX_W + 2)];

    // Current tile and apron limits, relative to row start
    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_W;
    const int xEnd = xStart + MINMAX_W - 1;
    const int xReadPos = xStart + tx - WARP_SIZE;
    const int xWritePos = xStart + tx;
    const int xEndClamped = min(xEnd, width - 1);
    int memWid = MINMAX_W + 2;

    int memPos0 = tx - WARP_SIZE + 1;
    int yq = 0;
    unsigned int output = 0;

    for (int y = 0; y < 32 + 2; y++) {
        output >>= 1;
        int memPos = yq * memWid + memPos0;
        int yp = 32 * blockIdx.y + y - 1;
        yp = max(yp, 0);
        yp = min(yp, height - 1);
        int readStart = yp * pitch;

        if (tx >= (WARP_SIZE - 1)) {
            data1[memPos] = (xReadPos >= 0 && xReadPos < width) ? d_Data1[readStart + xReadPos] : 0;
            data2[memPos] = (xReadPos >= 0 && xReadPos < width) ? d_Data2[readStart + xReadPos] : 0;
            data3[memPos] = (xReadPos >= 0 && xReadPos < width) ? d_Data3[readStart + xReadPos] : 0;
        }
        __syncthreads();

        int memPos2 = yq * memWid + tx;
        if (y > 1 && tx < memWid) {
            // Compute min and max for all data dimensions simultaneously
            float min1 = fminf(data1[memPos0], data1[memPos2]);
            float min2 = fminf(data2[memPos0], data2[memPos2]);
            float min3 = fminf(data3[memPos0], data3[memPos2]);
            float max1 = fmaxf(data1[memPos0], data1[memPos2]);
            float max2 = fmaxf(data2[memPos0], data2[memPos2]);
            float max3 = fmaxf(data3[memPos0], data3[memPos2]);

            ymin[tx] = fminf(fminf(min1, min2), min3);
            ymax[tx] = fmaxf(fmaxf(max1, max2), max3);
        }
        __syncthreads();

        if (y > 1 && tx < MINMAX_W && xWritePos <= xEndClamped) {
            float minv = fminf(ymin[tx], d_Threshold[1]);
            float maxv = fmaxf(ymax[tx], d_Threshold[0]);
            if (data2[memPos0 + 1] < minv || data2[memPos0 + 1] > maxv) {
                output |= 0x80000000;
            }
        }
        __syncthreads();

        memPos0 = memPos2;
        yq = (yq < 2 ? yq + 1 : 0);
    }

    if (tx < MINMAX_W && xWritePos < width) {
        int writeStart = blockIdx.y * pitch + xWritePos;
        d_Result[writeStart] = output;
    }
}