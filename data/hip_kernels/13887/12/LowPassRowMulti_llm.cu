#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LowPassRowMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    __shared__ float data[CONVROW_W + 2*RADIUS];
    const int tx = threadIdx.x;
    const int block = blockIdx.x/(NUM_SCALES+3);
    const int scale = blockIdx.x - (NUM_SCALES+3)*block;
    const int xout = block*CONVROW_W + tx;
    const int loadPos = xout - RADIUS;
    const int yptr = blockIdx.y*pitch;
    const int writePos = yptr + height*pitch*scale + xout;
    float *kernel = d_Kernel + scale*16;
    
    // Load data with boundary check using max and min 
    data[tx] = d_Data[yptr + max(0, min(loadPos, width - 1))];
    __syncthreads();

    // Perform the convolution only for valid xout
    if (xout < width && tx < CONVROW_W)
    {
        float result = 0.0f;
        // Unroll the loop for better performance
        #pragma unroll
        for (int i = 0; i < 4; ++i)
            result += (data[tx + i] + data[tx + 8 - i]) * kernel[i];
        result += data[tx + 4] * kernel[4];
        
        d_Result[writePos] = result;
    }
    __syncthreads();
}
```
