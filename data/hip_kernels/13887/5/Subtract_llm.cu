#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim.x instead of SUBTRACT_W for better compatibility
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Use blockDim.y instead of SUBTRACT_H for better compatibility
    int p = y * pitch + x;
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // Remove __syncthreads(); not necessary here, because there's no shared data
}