#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexFill(float *tensor, long* stride, float *index, long src_nDim, int dim, long idx_size, long tensor_size, long size_dim, float val) {
    // Calculate thread index
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    // Calculate flat size
    long flat_size = tensor_size / idx_size;

    // Check if within bounds
    if (thread_idx < flat_size) {
        long coeff = 0;
        
        // Main loop for index filling
        for (int i = 0; i < idx_size; i++) {
            int leftover = thread_idx;
            int srcIdx = 0;

            // Calculate source index efficiently
            for (int d = 0; d < src_nDim; d++) {
                coeff = leftover / ((d != dim) ? stride[d] : (stride[d] / size_dim));
                leftover -= coeff * ((d != dim) ? stride[d] : (stride[d] / size_dim));
                srcIdx += coeff * stride[d];
            }

            // Update tensor value
            tensor[srcIdx + (long)((index[i]) - 1) * stride[dim]] = val;
        }
    }
}