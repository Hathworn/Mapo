#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexSelect_contiguous(float *tensor, float *src, long stride, float *index, long idxSize) {
    // Constants
    const int VT = 4;
    const int WARP_SIZE = 32;
    const int MAX_DIM_SIZE = 65535;

    // Calculate index using block and thread index
    int idx = blockIdx.x * blockDim.y + threadIdx.y;
    int startIdx = threadIdx.x + blockIdx.y * VT * WARP_SIZE;

    // Loop through idx in steps to handle larger inputs
    while (idx < idxSize) {
        int start = startIdx;
        // Loop through startIdx to handle larger strides
        while (start < stride) {
            const long srcIdx = ((long)index[idx] - 1) * stride;
            const long targetIdx = idx * stride;

            // Unrolled loop to handle multiple elements per thread
            #pragma unroll
            for (int i = 0; i < VT; i++) {
                const int featureIdx = start + i * WARP_SIZE;
                // Ensure we are within stride limits
                if (featureIdx < stride) {
                    tensor[targetIdx + featureIdx] = src[srcIdx + featureIdx];
                }
            }
            start += VT * WARP_SIZE * MAX_DIM_SIZE;
        }
        idx += blockDim.y * MAX_DIM_SIZE;
    }
}