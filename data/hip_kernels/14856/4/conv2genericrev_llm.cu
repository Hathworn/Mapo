#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv2genericrev(float *input, float *kernel, float *output, int input_n, int input_h, int input_w, int kernel_n, int kernel_h, int kernel_w, float alpha, int stride_h, int stride_w)
{
    // Output dimensions
    int output_h = input_h - (kernel_h - 1) * stride_h;
    int output_w = input_w - (kernel_w - 1) * stride_w;

    // Thread indices
    int kk = blockIdx.x;
    int ii = blockIdx.y;

    // Batch and kernel id
    int batch = threadIdx.z;
    int kid = threadIdx.x;
    int nkids = blockDim.x;

    // Linear thread ID for kernel calculations
    int tid = kid + batch * blockDim.x;
    int nthreads = blockDim.x * blockDim.z;

    // Output index and shared memory pointer
    output = output + (kk * input_n + ii) * output_h * output_w;
    extern __shared__ float shared_output[]; // Use dynamic shared memory

    // Initialize shared memory to zero
    float *output_s = shared_output + tid * output_w;
    for (int i = 0; i < output_w; ++i)
        output_s[i] = 0.0;

    // Convolution loop
    for (int yy = threadIdx.y; yy < output_h; yy += blockDim.y) {
        float *output_p = output_s;
        for (int xx=0; xx < output_w; xx++) {
            float *input_p = input + (ii + batch * input_n) * input_h * input_w + yy * stride_h * input_w + xx * stride_w;
            float *kernel_p = kernel + (kk + batch * kernel_n) * kernel_w * kernel_h;
            
            float sum = 0;
            for (int ky = 0; ky < kernel_h; ky++) {
                for (int kx = kid; kx < kernel_w; kx += nkids) {
                    sum += input_p[kx] * kernel_p[kx];
                }
                input_p += input_w;
                kernel_p += kernel_w;
            }
            output_p[xx] += sum;
        }
    }
    __syncthreads();

    // Reduce outputs and write back
    if (yy == 0) {
        // Reduce outputs for all threads
        for (int k = 1; k < nthreads; k++) {
            for (int i = tid; i < output_w * output_h; i += nthreads) {
                shared_output[i] += shared_output[k * output_h * output_w + i];
            }
        }
        __syncthreads();

        // Add existing output, and write back
        for (int i = tid; i < output_w * output_h; i += nthreads) {
            output[i] += alpha * shared_output[i];
        }
    }
}