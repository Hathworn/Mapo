#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;

        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithReplacement(hiprandStateMtgp32* state, int totalSamples, float* dest, long distributions, int categories, float* normDistPrefixSum) {

    // Use shared memory for random states to reduce global memory accesses
    __shared__ float sharedRandom[32];
    
    for (long curDist = blockIdx.x; curDist < distributions; curDist += gridDim.x) {
        
        for (int sampleBase = 0; sampleBase < totalSamples; sampleBase += blockDim.y) {
            
            int sample = sampleBase + threadIdx.y;
            
            // Each thread in a warp generates its own random number, saved in shared memory
            sharedRandom[threadIdx.y] = hiprand_uniform(&state[blockIdx.x]);
            __syncthreads(); // Ensure all random numbers are generated before proceeding

            if (threadIdx.x == 0 && sample < totalSamples) {
                float r = sharedRandom[threadIdx.y]; // Read random number from shared memory

                int choice = binarySearchForMultinomial(
                    normDistPrefixSum + curDist * categories,
                    categories,
                    r);

                dest[curDist * totalSamples + sample] = (float) choice + 1.0f;
            }
        }
    }
}