#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;
    while (end - start > 0) {
        int mid = start + (end - start) / 2;
        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }
    if (start == size) {
        start = 0;
    }
    return start;
}

__global__ void sampleMultinomialWithoutReplacement(hiprandStateMtgp32* state, int totalSamples, int sample, float* dest, long distributions, int categories, float* origDist, float* normDistPrefixSum) {
    long curDistBase = blockIdx.x * blockDim.y;
    long curDist = curDistBase + threadIdx.y;

    if (curDist < distributions) { // Only process valid distributions
        float r = hiprand_uniform(&state[blockIdx.x]);

        if (threadIdx.x == 0) {
            int choice = binarySearchForMultinomial(normDistPrefixSum + curDist * categories, categories, r);
            dest[curDist * totalSamples + sample] = (float) choice + 1.0f;
            origDist[curDist * categories + choice] = 0.0f;
        }
    }
}
```
