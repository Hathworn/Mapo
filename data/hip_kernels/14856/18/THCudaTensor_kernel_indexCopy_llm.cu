#include "hip/hip_runtime.h"
```
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexCopy( float *res, float *src, long* res_stride, float *index, long res_nDim, int dim, long idx_size, long src_size, long size_dim )
{
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    
    // Calculate the flat size outside the loop
    long flat_size = src_size / idx_size;

    if (thread_idx < flat_size)
    {
        long targetIdxBase = 0;
        long resIdxBase = 0;
        int leftover = thread_idx;

        for (int d = 0; d < res_nDim; ++d)
        {
            if (d != dim)
            {
                long stride_d = (d < dim) ? (res_stride[d] / size_dim) : res_stride[d];
                long coeff = leftover / stride_d;
                leftover -= coeff * stride_d;
                targetIdxBase += coeff * stride_d * ((d < dim) ? idx_size : 1);
                resIdxBase += coeff * res_stride[d];
            }
        }

        // Use index out of the loop, reducing redundant computations
        for (int i = 0; i < idx_size; ++i)
        {
            long targetIdx = targetIdxBase + i * res_stride[dim];
            long indexOffset = ((long)(index[i]) - 1) * res_stride[dim];
            res[resIdxBase + indexOffset] = src[targetIdx];
        }
    }
}