#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexAdd(float *res, float *src, long *res_stride, float *index, long res_nDim, int dim, long idx_size, long src_size, long size_dim)
{
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    long flat_size = src_size / idx_size;

    if (thread_idx < flat_size)
    {
        long coeff = 0;
        long targetIdx = 0;
        long resIdx = 0;
        int leftover = thread_idx;

        // Precompute common strides to save redundant calculations inside the loop
        long stride_precompute[dim];

        for (int d = 0; d < dim; d++)
        {
            stride_precompute[d] = res_stride[d] / size_dim;
        }

        for (int i = 0; i < idx_size; i++)
        {
            for (int d = 0; d < res_nDim; d++)
            {
                if (d < dim)
                {
                    coeff = leftover / stride_precompute[d];
                    leftover -= coeff * stride_precompute[d];
                    targetIdx += coeff * stride_precompute[d] * idx_size;
                    resIdx += coeff * res_stride[d];
                }
                else if (d > dim)
                {
                    coeff = leftover / res_stride[d];
                    leftover -= coeff * res_stride[d];
                    targetIdx += coeff * res_stride[d];
                    resIdx += coeff * res_stride[d];
                }
            }
            // Improved atomic operation indexing
            atomicAdd(&res[resIdx + ((long)(index[i]) - 1) * res_stride[dim]], src[targetIdx + i * res_stride[dim]]);
        }
    }
}