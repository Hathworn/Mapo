#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_renorm(float *data, const float value, const long size, const float maxnorm)
{
    __shared__ float buffer[32];
    const long tx = threadIdx.x;
    const long bx = blockIdx.x;
    const long step = blockDim.x;
    float *row = data + size * bx;

    // Initialize buffer with zero
    buffer[tx] = 0.0f;

    // Calculate partial norms in parallel
    for (long i = tx; i < size; i += step)
    {
        buffer[tx] += powf(fabsf(row[i]), value);
    }

    // Efficient parallel reduction using shared memory
    __syncthreads(); // Ensure all partial sums are done
    for (unsigned int stride = step / 2; stride > 0; stride >>= 1)
    {
        if (tx < stride)
        {
            buffer[tx] += buffer[tx + stride];
        }
        __syncthreads(); // Synchronize to ensure correct reduction
    }

    // Normalize the row based on the calculated norm
    if (tx == 0)
    {
        float norm = powf(buffer[0], 1.0f / value);
        if (norm > maxnorm)
        {
            norm = maxnorm / (norm + 1e-7f);
            buffer[0] = norm;
        }
        else
        {
            buffer[0] = 1.0f; // No renorm required
        }
    }

    // Apply renormalization if needed
    __syncthreads(); // Ensure buffer[0] is ready
    for (long i = tx; i < size; i += step)
    {
        row[i] *= buffer[0];
    }
}