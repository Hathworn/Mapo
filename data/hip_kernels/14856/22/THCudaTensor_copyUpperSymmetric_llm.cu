#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_copyUpperSymmetric(float *input, int n, int len)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop unrolling to improve performance
    for (int i = idx; i < len; i += gridDim.x * blockDim.x) {
        const int r = i % n;
        const int c = i / n;
        if (r > c) {
            input[i] = input[r * n + c];
        }
    }
}