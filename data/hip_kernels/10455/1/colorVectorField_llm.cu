#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void colorVectorField( float3 *colors, float3 *colorMap, float2 *__restrict__ field, dim3 blocks, unsigned int simWidth, unsigned int simHeight)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= simWidth || y >= simHeight) return; // Bounds check for thread
    
    int quadIdx = x + simWidth * y;
    
    // Print debug information for specific location
    if (x == simWidth / 2 + 10 && y == simWidth / 2 + 10)
        printf("coloring vector field values on the order of: %f, %f\n", field[quadIdx].x, field[quadIdx].y);
    
    float mag = field[quadIdx].x; // Use precomputed magnitude
    int map = (int)(mag / 0.004 * 256);
    map = min(max(map, 0), 255); // Clamp values between 0 and 255

    // Assign colors based on color map and conditions
    float3 color;
    if (field[quadIdx].x == 0)
        color = make_float3(0.0, 0.6, 0.2);
    else if (mag < 0)
        color = make_float3(0.0, 0.3, 0.7);
    else
        color = colorMap[map];
    
    for (int i = 0; i < 4; i++) {
        colors[4 * quadIdx + i] = color;
    }
}