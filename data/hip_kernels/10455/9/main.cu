#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "updateVel.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float2 __restrict__ *oldVel = NULL;
hipMalloc(&oldVel, XSIZE*YSIZE);
float2 __restrict__ *newVel = NULL;
hipMalloc(&newVel, XSIZE*YSIZE);
unsigned int simWidth = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
updateVel<<<gridBlock, threadBlock>>>(oldVel,newVel,simWidth);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
updateVel<<<gridBlock, threadBlock>>>(oldVel,newVel,simWidth);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
updateVel<<<gridBlock, threadBlock>>>(oldVel,newVel,simWidth);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}