#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateVel(float2 *__restrict__ oldVel, float2 *__restrict__ newVel, unsigned int simWidth)
{
    // Compute global index with shared memory optimization
    unsigned int idx = blockIdx.y * blockDim.y * simWidth + blockIdx.x * blockDim.x + threadIdx.y * simWidth + threadIdx.x;

    // Directly copy velocity values using computed index
    oldVel[idx] = newVel[idx];
}