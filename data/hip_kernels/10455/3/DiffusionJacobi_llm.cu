#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float2 JacobiFieldInstance(float2 Top, float2 Left, float2 Bot, float2 Right, float Alpha, float2 Val)
{
    float2 res;
    res.x = (Top.x + Left.x + Bot.x + Right.x + Alpha * Val.x) / (4 + Alpha);
    res.y = (Top.y + Left.y + Bot.y + Right.y + Alpha * Val.y) / (4 + Alpha);
    return res;
}

__global__ void DiffusionJacobi(float2 *__restrict__ positions, float2 *__restrict__ oldVel, float2 *__restrict__ newVel, float dt, float dr, float viscosity, unsigned int simWidth, unsigned int simHeight)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x == 0 || y == 0 || x == simWidth-1 || y == simHeight-1) {
        return;  // Skip boundary conditions early for better performance
    }

    float2 Vel = oldVel[y * simWidth + x];
    float alpha = dr * dr / (viscosity * dt);

    float2 TVel = oldVel[(y - 1) * simWidth + x];
    float2 LVel = oldVel[y * simWidth + x - 1];
    float2 BVel = oldVel[(y + 1) * simWidth + x];
    float2 RVel = oldVel[y * simWidth + x + 1];

    newVel[y * simWidth + x] = JacobiFieldInstance(TVel, LVel, BVel, RVel, alpha, Vel);
}