#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "DiffusionJacobi_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float2 __restrict__ *positions = NULL;
hipMalloc(&positions, XSIZE*YSIZE);
float2 __restrict__ *oldVel = NULL;
hipMalloc(&oldVel, XSIZE*YSIZE);
float2 __restrict__ *newVel = NULL;
hipMalloc(&newVel, XSIZE*YSIZE);
float dt = 1;
float dr = 1;
float viscosity = 1;
unsigned int simWidth = 1;
unsigned int simHeight = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
DiffusionJacobi<<<gridBlock, threadBlock>>>(positions,oldVel,newVel,dt,dr,viscosity,simWidth,simHeight);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
DiffusionJacobi<<<gridBlock, threadBlock>>>(positions,oldVel,newVel,dt,dr,viscosity,simWidth,simHeight);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
DiffusionJacobi<<<gridBlock, threadBlock>>>(positions,oldVel,newVel,dt,dr,viscosity,simWidth,simHeight);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}