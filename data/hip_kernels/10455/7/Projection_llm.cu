#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Projection(float2 *__restrict__ newVel, float2 *__restrict__ gradPressure, unsigned int simWidth)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int index = y * simWidth + x; // Precompute index for reusability

    // Use shared memory for coalesced global memory access (optional, if beneficial)
    extern __shared__ float2 sharedMem[];
    sharedMem[threadIdx.y * blockDim.x + threadIdx.x] = gradPressure[index];
    __syncthreads();

    // Compute newVel with optimized global memory access
    newVel[index].x -= sharedMem[threadIdx.y * blockDim.x + threadIdx.x].x;
    newVel[index].y -= sharedMem[threadIdx.y * blockDim.x + threadIdx.x].y;
}