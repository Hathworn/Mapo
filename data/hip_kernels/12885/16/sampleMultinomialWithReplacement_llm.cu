#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;
        
        // Access dist mid only once and compare
        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        start = 0;  // Return the first element if outside range
    }

    return start;
}

__global__ void sampleMultinomialWithReplacement(hiprandStateMtgp32* state, int totalSamples, float* dest, long distributions, int categories, float* normDistPrefixSum) {
    for (long curDist = blockIdx.x; curDist < distributions; curDist += gridDim.x) {
        for (int sampleBase = 0; sampleBase < totalSamples; sampleBase += blockDim.y) {
            int sample = sampleBase + threadIdx.y;

            // Use shared memory to reduce redundant accesses or computations
            __shared__ float r[1];
            if (threadIdx.x == 0 && sample < totalSamples) {
                r[0] = hiprand_uniform(&state[blockIdx.x]);
            }
            __syncthreads();

            if (threadIdx.x == 0 && sample < totalSamples) {
                // Perform binary search to find the correct category
                int choice = binarySearchForMultinomial(normDistPrefixSum + curDist * categories, categories, r[0]);
                
                // Store the result with added 1.0 offset for 1-based index
                dest[curDist * totalSamples + sample] = (float) choice + 1.0f;
            }
        }
    }
}