#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_renorm(float *data, const float value, const long size, const float maxnorm)
{
    extern __shared__ float buffer[]; // Use dynamic shared memory
    long tx = threadIdx.x;
    long bx = blockIdx.x;
    long step = blockDim.x;
    float *row = data + size * bx;

    float sum = 0.0f;

    // Compute partial sums
    for (long i = tx; i < size; i += step)
    {
        sum += pow(fabsf(row[i]), value);
    }
    buffer[tx] = sum;

    // Reduce sums
    __syncthreads();
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
    {
        if (tx < stride)
        {
            buffer[tx] += buffer[tx + stride];
        }
        __syncthreads();
    }

    // Clip norms and renormalize if needed
    if (tx == 0)
    {
        float norm = pow(buffer[0], 1 / value);
        if (norm > maxnorm)
        {
            norm = maxnorm / (norm + 1e-7);
            buffer[0] = norm;
        }
        else
        {
            buffer[0] = 1.0f; // No renormalization needed
        }
    }

    // Ensure all threads are synchronized
    __syncthreads();

    float normFactor = buffer[0];
    for (long i = tx; i < size; i += step)
    {
        row[i] *= normFactor;
    }
}