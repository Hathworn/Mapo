#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;
        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        // No probability mass or precision problems; return the first element
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithoutReplacement(hiprandStateMtgp32* state, int totalSamples, int sample, float* dest, long distributions, int categories, float* origDist, float* normDistPrefixSum) {
    // Precompute distribution index limits
    long maxDistBase = gridDim.x * blockDim.y;
    
    // Loop over distributions handling within the block
    for (long curDistBase = blockIdx.x * blockDim.y; curDistBase < distributions; curDistBase += maxDistBase) {
        long curDist = curDistBase + threadIdx.y;

        // All threads must participate in this for random number generation
        float r = hiprand_uniform(&state[blockIdx.x]);

        if (threadIdx.x == 0 && curDist < distributions) {
            // Binary search for the sampled category
            int choice = binarySearchForMultinomial(normDistPrefixSum + curDist * categories, categories, r);

            // Store the sampled category, note 1-based index
            dest[curDist * totalSamples + sample] = (float) choice + 1.0f;

            // Update probability for non-replacement by setting it to zero
            origDist[curDist * categories + choice] = 0.0f;
        }
    }
}