#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmul(double *a, double *b, double *c, int n)
{
    // Shared memory for sub-matrices
    extern __shared__ double shared_mem[];

    // Index for shared memory
    double* As = &shared_mem[0];
    double* Bs = &shared_mem[blockDim.x * blockDim.y];

    // Get global thread ID
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;

    // Initialize the sum to zero
    double sum = 0.0;

    // Collaboratively load tiles into shared memory
    for (int tileIdx = 0; tileIdx < (n + blockDim.x - 1) / blockDim.x; ++tileIdx) {
        // Load elements of a into shared memory
        if (Row < n && (tileIdx * blockDim.x + threadIdx.x) < n)
            As[threadIdx.y * blockDim.x + threadIdx.x] = a[Row * n + tileIdx * blockDim.x + threadIdx.x];
        else
            As[threadIdx.y * blockDim.x + threadIdx.x] = 0.0;

        // Load elements of b into shared memory
        if (Col < n && (tileIdx * blockDim.y + threadIdx.y) < n)
            Bs[threadIdx.y * blockDim.x + threadIdx.x] = b[(tileIdx * blockDim.y + threadIdx.y) * n + Col];
        else
            Bs[threadIdx.y * blockDim.x + threadIdx.x] = 0.0;

        // Synchronize threads to ensure complete loading
        __syncthreads();

        // Compute partial matrix multiplication for the tile
        for (int k = 0; k < blockDim.x; ++k) {
            sum += As[threadIdx.y * blockDim.x + k] * Bs[k * blockDim.x + threadIdx.x];
        }

        // Synchronize before loading the next tile
        __syncthreads();
    }

    // Write result to global memory
    if ((Col < n) && (Row < n)) {
        c[Row * n + Col] = sum;
    }
}
```
