#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Using shared memory for better memory access performance
    extern __shared__ double shmem[];
    double *sh_a = shmem;                    // Shared memory for a
    double *sh_b = shmem + blockDim.x;       // Shared memory for b

    // Get our global thread ID and local thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Ensure global thread ID is within bounds and load inputs to shared memory
    if (id < n) {
        sh_a[tid] = a[id];
        sh_b[tid] = b[id];
    }

    __syncthreads(); // Synchronize threads

    // Calculate and write the result if within bounds
    if (id < n) {
        c[id] = sh_a[tid] + sh_b[tid];
    }
}