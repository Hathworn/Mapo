#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized HIP kernel for vector addition

__global__ void vecAdd(float *a, float *b, float *c, int n)
{
    // Calculate unique global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Process multiple elements per thread to improve memory access patterns
    int stride = blockDim.x * gridDim.x;
    for (int i = id; i < n; i += stride) {
        c[i] = a[i] + b[i];
    }
}