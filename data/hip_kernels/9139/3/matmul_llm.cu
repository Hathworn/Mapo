#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmul(double *a, double *b, double *c, int n)
{
    // Cache block's shared memory space
    extern __shared__ double shared[];

    // Define sub-matrix dimensions
    int tile_size = blockDim.x;

    // Calculate matrix indices
    int Row = blockIdx.y * tile_size + threadIdx.y;
    int Col = blockIdx.x * tile_size + threadIdx.x;

    // Initialize sum accumulator
    double sum = 0.0;

    // Iterate over sub-matrix blocks
    for(int i = 0; i < n; i += tile_size) {
        // Load elements into shared memory
        shared[threadIdx.y * tile_size + threadIdx.x] = (Row < n && (i + threadIdx.x) < n) ? a[Row * n + i + threadIdx.x] : 0.0;
        shared[tile_size * tile_size + threadIdx.y * tile_size + threadIdx.x] = ((i + threadIdx.y) < n && Col < n) ? b[(i + threadIdx.y) * n + Col] : 0.0;

        // Synchronize within block
        __syncthreads();

        // Multiply and accumulate sub-matrix
        for(int j = 0; j < tile_size; j++) {
            sum += shared[threadIdx.y * tile_size + j] * shared[tile_size * tile_size + j * tile_size + threadIdx.x];
        }

        // Synchronize within block
        __syncthreads();
    }

    // Write result to global memory
    if(Row < n && Col < n) {
        c[Row * n + Col] = sum;
    }
}