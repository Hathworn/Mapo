#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiply_kernel(float* A, float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Only compute values within the array bounds
    if (i < N) {
        // Simple element-wise multiplication
        C[i] = A[i] * B[i];
    }
}