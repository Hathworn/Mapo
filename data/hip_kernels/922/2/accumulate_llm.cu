#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
    long long int x;
};

__global__ void accumulate(Edge* edge_list, bool* cross_edges, int* indices, int e) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure no overwriting occurs in parallel writing
    if (id < e && cross_edges[id]) {
        Edge temp = edge_list[id];
        atomicExch(&edge_list[indices[id]].x, temp.x);
    }
}