#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) ((a) > (b) ? (a) : (b))
#define min(a, b) ((a) < (b) ? (a) : (b))

struct Edge{
    long long int x;
};

__global__ void process_cross_edges(int* parent, Edge* edge_list, int e, bool* flag, bool* cross_edges) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ bool block_flag;
    
    if (threadIdx.x == 0) {
        block_flag = false; // Initialize block shared flag
    }
    __syncthreads();

    if (id < e && cross_edges[id]) {
        long long int x = edge_list[id].x;
        int v = (int)x & 0xFFFFFFFF;
        int u = (int)(x >> 32);

        int parent_u = parent[u];
        int parent_v = parent[v];

        if (parent_u != parent_v) { // Check if nodes have different parents
            int mn = min(parent_u, parent_v);
            int mx = max(parent_u, parent_v);
            parent[mx] = mn; // Union operation
            block_flag = true; // Set block flag if union occurs
        } else {
            cross_edges[id] = false;
        }
    }
    __syncthreads();

    if (threadIdx.x == 0 && block_flag) {
        *flag = true; // If any union occurred, set global flag
    }
}