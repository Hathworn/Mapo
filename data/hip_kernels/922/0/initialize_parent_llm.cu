#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
    long long int x;
};

__global__ void initialize_parent(int* parent, int n){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a conditional statement to ensure accessing valid indices
    if(id < n){
        parent[id] = id;
    }
}