#include "hip/hip_runtime.h"
```cuda
#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
long long int x;
};

__global__ void pointer_jumping(int* parent, int n, bool* flag) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (id < n) {
        int parent_id = parent[id];
        int grandparent_id = parent[parent_id];

        // Use warp-wide operations to reduce the need for __syncthreads()
        bool local_flag = (parent_id != grandparent_id);
        if (local_flag) {
            parent[id] = grandparent_id;
        }

        // Use atomic operations to avoid repeated checks by threads at different warps
        if (local_flag && __syncthreads_or(local_flag)) {
            *flag = true;
        }
    }
}