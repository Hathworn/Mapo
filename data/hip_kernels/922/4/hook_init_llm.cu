#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
    long long int x;
};

__global__ void hook_init(int* parent, Edge* edge_list, int e){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within bounds and avoid shared variables
    if(id < e){
        long long int x = edge_list[id].x;
        int v = (int) x & 0xFFFFFFFF;
        int u = (int) (x >> 32);

        // Directly calculate min and max using ternary operator
        int mx = u > v ? u : v;
        int mn = u <= v ? u : v;
        
        parent[mx] = mn;
    }
}