#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) ((a) > (b) ? (a) : (b))
#define min(a, b) ((a) < (b) ? (a) : (b))

struct Edge {
    long long int x;
};

__global__ void hook_odd(int* parent, Edge* edge_list, int e, bool* flag, bool* active_edges){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Declare shared block flag with initial false
    __shared__ bool block_flag;
    if (threadIdx.x == 0) block_flag = false;
    __syncthreads();

    // Process active edges only
    if (id < e && active_edges[id]) {
        long long int x = edge_list[id].x;
        int v = (int) x & 0xFFFFFFFF;
        int u = (int) (x >> 32);

        int parent_u = parent[u];
        int parent_v = parent[v];

        // Use min and max functions directly for simplicity
        int mx = max(parent_u, parent_v);
        int mn = parent_u + parent_v - mx;

        // Update active status and flag if necessary
        if (parent_u == parent_v) {
            active_edges[id] = false;
        } else {
            parent[mx] = mn;
            block_flag = true;
        }
    }
    __syncthreads();

    // Update the global flag only once per block
    if (threadIdx.x == 0 && block_flag) {
        *flag = true;
    }
}