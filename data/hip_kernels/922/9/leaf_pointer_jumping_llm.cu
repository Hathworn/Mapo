#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
    long long int x;
};

__global__ void leaf_pointer_jumping(int* parent, int* vertex_state, int n) {
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check bounds and vertex state
    if (id < n && vertex_state[id] == 1) {
        // Efficiently update parent pointer
        int parent_id = parent[id];
        parent[id] = parent[parent_id];
    }
}