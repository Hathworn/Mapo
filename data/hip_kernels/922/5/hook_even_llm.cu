#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
    long long int x;
};

__global__ void hook_even(int* parent, Edge* edge_list, int e, bool* flag, bool* active_edges) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int id = bid * blockDim.x + tid;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory to reduce global memory access frequency
    __shared__ bool block_flag;
    if (tid == 0) block_flag = false;
    __syncthreads();

    for (int i = id; i < e; i += stride) {
        if (active_edges[i]) {
            long long int x = edge_list[i].x;
            int v = (int)x & 0xFFFFFFFF;
            int u = (int)(x >> 32);

            int parent_u = parent[u];
            int parent_v = parent[v];

            int mx = max(parent_u, parent_v);
            int mn = parent_u + parent_v - mx;

            if (parent_u == parent_v) {
                active_edges[i] = false;
            } else {
                parent[mn] = mx;
                block_flag = true;
            }
        }
    }
    __syncthreads();

    if (tid == 0 && block_flag) {
        *flag = true;
    }
}