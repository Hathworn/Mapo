#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
    long long int x;
};

__global__ void update_states(int* parent, int* vertex_state, int n) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread index is within bounds
    if (id < n) {
        // Efficient conditional assignment using ternary operator
        vertex_state[id] = (parent[id] == id) ? 0 : 1;
    }
}