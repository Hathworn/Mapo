#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
long long int x;
};

__global__ void root_pointer_jumping(int* parent, int* vertex_state, int n, bool* flag) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    int id = bid * blockDim.x + tid;
    int parent_id, grandparent_id;
    __shared__ bool block_flag;
    if (tid == 0) block_flag = false;
    __syncthreads();

    if (id < n && vertex_state[id] == 0) {  // Combined two if conditions
        parent_id = parent[id];
        grandparent_id = parent[parent_id];
        if (parent_id != grandparent_id) {
            parent[id] = grandparent_id;
            block_flag = true;
        } else {
            vertex_state[id] = -1;
        }
    }
    
    __syncthreads();  // Ensure all threads update block_flag before any thread writes to global flag

    if (tid == 0 && block_flag) *flag = true;
}