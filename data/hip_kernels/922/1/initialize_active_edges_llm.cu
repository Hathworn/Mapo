#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define max(a, b) a > b ? a : b
#define min(a, b) a < b ? a : b

struct Edge{
    long long int x;
};

__global__ void initialize_active_edges(bool* active_edges, int e) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check within bounds and set each edge as active
    if (id < e) {
        active_edges[id] = true;
    }
}