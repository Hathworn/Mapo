#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ap_multiplication(float * values, int * indices, float * r, float * p_sum, int size)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (index < size) {
        float sum = 0.0f; // Use local variable to accumulate sum
        for (int i = 0; i < 3; i++) {
            sum += values[3 * index + i] * r[indices[3 * index + i]];
        }
        p_sum[index] = sum; // Store the result back
    }
}