#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GaussianEliminationGlobal(const int clusterSize, float *x, const float *diagonal_values, const float *non_diagonal_values, float *y, const int size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int gi = index * clusterSize;

    // Use shared memory for faster access, as matrix size is now manageable
    __shared__ float matrix[180][180];

    // Initialize matrix
    for (int i = gi; i < gi + clusterSize; ++i)
    {
        for (int j = gi; j < gi + clusterSize; ++j)
        {
            matrix[i][j] = 0;
        }
        matrix[i][i] = diagonal_values[i];
    }
    // Fill non-diagonal values
    for (int i = gi; i < gi + clusterSize - 1; ++i)
    {
        matrix[i][i+1] = non_diagonal_values[2*i+1];
        matrix[i+1][i] = non_diagonal_values[2*i+2];
    }

    // Forward elimination to triangular form
    for (int i = gi; i < gi + clusterSize; ++i)
    {
        for (int j = i+1; j < gi + clusterSize; ++j)
        {
            // Calculate ratio for elimination
            float ratio = matrix[j][i] / matrix[i][i];
            for (int k = gi; k < gi + clusterSize; ++k)
            {
                matrix[j][k] -= matrix[i][k] * ratio;
            }
            y[j] -= y[i] * ratio;
        }
    }
    __syncthreads();

    // Back substitution
    for (int i = gi + clusterSize - 1; i >= gi; --i)
    {
        float current = 0;
        for (int j = i+1; j < gi + clusterSize; ++j)
        {
            current += matrix[i][j] * x[j];
        }
        x[i] = (y[i] - current) / matrix[i][i];
    }
}