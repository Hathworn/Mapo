#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

char* concat(char *s1, char *s2);

__global__ void r_calculation(float* a, int* indeces, float* b, float* x, float* r, int size) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory for storing intermediate values to improve memory access speed
    __shared__ float shared_a[3];
    __shared__ int shared_indices[3];
    __shared__ float shared_x[3];

    if (index < size) {
        float sum = 0;

        // Unrolling the loop to reduce overhead and increase parallel execution
        #pragma unroll
        for (int i = 0; i < 3; i++) {
            shared_a[i] = a[3 * index + i];
            shared_indices[i] = indeces[3 * index + i];
            shared_x[i] = x[shared_indices[i]];
            sum += shared_a[i] * shared_x[i];
        }

        r[index] = b[index] - sum;
    }
}