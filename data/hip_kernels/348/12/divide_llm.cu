#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void divide(float *x, float* y, float* out, const int size)
{
    // Calculate the global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate the stride

    // Loop over elements with stride step
    for (int i = index; i < size; i += stride) 
    {
        out[i] = x[i] / y[i]; // Perform division
    }
}