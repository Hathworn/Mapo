#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GaussianEliminationShared(const int clusterSize,float *x, const float *diagonal_values , const float *non_diagonal_values ,float *y )
{
    const int index = blockIdx.x;
    
    __shared__ float shared_m[9][9]; // size of cluster

    // Initialize shared memory with diagonal and non-diagonal values
    for(int i = 0; i < clusterSize; ++i)
    {
        for(int j = 0; j < clusterSize; ++j)
        {
            shared_m[i][j] = (i == j) ? diagonal_values[clusterSize * index + i] : 0;
        }
    }
    
    for(int i = 0; i < clusterSize - 1; ++i)
    {
        shared_m[i][i + 1] = non_diagonal_values[clusterSize * index * 2 + 2 * i + 1];
        shared_m[i + 1][i] = non_diagonal_values[clusterSize * index * 2 + 2 * i + 2];
    }

    // Triangle form transformation using Gaussian elimination
    for (int i = 0; i < clusterSize; ++i)
    {
        // For each row below the current one
        for (int j = i + 1; j < clusterSize; ++j)
        {
            // Calculate elimination ratio
            float ratio = shared_m[j][i] / shared_m[i][i];
            
            for(int k = i; k < clusterSize; ++k)
            {
                // Perform elimination on matrix
                shared_m[j][k] -= shared_m[i][k] * ratio;
            }
            
            // Perform elimination on coefficient vector
            y[clusterSize * index + j] -= y[clusterSize * index + i] * ratio;
        }
    }

    __syncthreads();

    // Back substitution to solve x
    for (int i = clusterSize - 1; i >= 0; --i)
    {
        float sum = 0.0f;
        
        for (int j = i + 1; j < clusterSize; ++j)
        {
            sum += shared_m[i][j] * x[clusterSize * index + j];
        }
        
        x[clusterSize * index + i] = (y[clusterSize * index + i] - sum) / shared_m[i][i];
    }
}