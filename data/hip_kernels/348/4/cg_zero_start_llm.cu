#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cg_zero_start(float* a, float* x, float* b, int size) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int local_index = threadIdx.x;
    
    __shared__ float shared_r_squared[1024];
    __shared__ float shared_p_sum[1024];
    
    // Initialize shared memory
    shared_r_squared[local_index] = 0.0f;
    shared_p_sum[local_index] = 0.0f;
    
    __syncthreads();

    if (index < size) {
        float local_b = b[index + 2 * blockIdx.x + 1];

        // Compute partial sums
        for (int i = 0; i < 3; i++) {
            shared_p_sum[local_index] += a[3 * index + i] * b[index + 2 * blockIdx.x + i];
        }

        __syncthreads();

        // Update shared memory for r_squared and p_sum
        shared_r_squared[local_index] = local_b * local_b;
        shared_p_sum[local_index] *= local_b;

        __syncthreads();

        // Reduction using shared memory for r_squared and p_sum
        for (unsigned int s = blockDim.x >> 1; s > 0; s >>= 1) {
            if (local_index < s) {
                shared_r_squared[local_index] += shared_r_squared[local_index + s];
                shared_p_sum[local_index] += shared_p_sum[local_index + s];
            }
            __syncthreads();
        }

        // Calculate alpha and update x
        if (local_index == 0) {
            float alpha = shared_r_squared[0] / shared_p_sum[0];
            x[index] += alpha * local_b;
        }
    }
}