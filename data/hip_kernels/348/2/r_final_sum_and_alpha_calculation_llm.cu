#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

char* concat(char *s1, char *s2);

__global__ void r_final_sum_and_alpha_calculation(float *r_squared, float *p_sum, int size) {
    int index = threadIdx.x;
    extern __shared__ float shared_mem[]; // Use dynamic shared memory
    float* shared_r_squared = shared_mem;
    float* shared_p_sum = shared_mem + blockDim.x;

    // Load input into shared memory
    float local_r_squared = (index < size) ? r_squared[index] : 0.0f;
    float local_p_sum = (index < size) ? p_sum[index] : 0.0f;

    shared_r_squared[index] = local_r_squared;
    shared_p_sum[index] = local_p_sum;

    __syncthreads();

    // Reduce within block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (index < s) {
            shared_r_squared[index] += shared_r_squared[index + s];
            shared_p_sum[index] += shared_p_sum[index + s];
        }
        __syncthreads();
    }

    // Write block result
    if (index == 0) {
        r_squared[blockIdx.x] = shared_r_squared[0] / shared_p_sum[0];
    }
}