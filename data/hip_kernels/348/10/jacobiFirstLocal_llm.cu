#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void jacobiFirstLocal(float *x, const float *diagonal_values, const float *non_diagonal_values, const int *indeces, const float *y, const int size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index < size)
    {
        float local_diagonal_value = diagonal_values[index];
        float local_non_diagonal_values[2] = {non_diagonal_values[2 * index], non_diagonal_values[2 * index + 1]};
        int local_indeces[2] = {indeces[2 * index], indeces[2 * index + 1]};
        float local_y = y[index];

        float error = 1.0f;
        float current_value = x[index];  // Use initial value of x for more efficient convergence
        float sum = 0.0f;

        while (fabsf(error) > 0.00001)
        {
            sum = local_non_diagonal_values[0] * x[local_indeces[0]] + local_non_diagonal_values[1] * x[local_indeces[1]];  // Pre-compute sum

            float new_value = (local_y - sum) / local_diagonal_value;  // Calculate new value
            error = new_value - current_value;  // Compute error
            current_value = new_value;  // Update current value

            x[index] = current_value;  // Write back current value
            __syncthreads();
        }
    }
}
```
