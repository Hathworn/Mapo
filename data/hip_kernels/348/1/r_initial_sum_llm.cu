#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

char* concat(char *s1, char *s2);

__global__ void r_initial_sum(float* a, int* indeces, float* x, float* r, float* r_squared, float* p_sum, int size)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int local_index = threadIdx.x;

    __shared__ float shared_r_squared[1024];
    __shared__ float shared_p_sum[1024];

    // Initialize shared memory
    float sum_p_val = 0;
    float sum_r_squared_val = 0;
    
    if (index < size)
    {
        // Compute partial sums directly into registers
        for (int i = 0; i < 3; i++)
        {
            sum_p_val += a[3 * index + i] * r[indeces[3 * index + i]];
        }
        sum_r_squared_val = r[index] * r[index];
        sum_p_val = sum_p_val * r[index];
    }
    
    // Store results in shared memory
    shared_r_squared[local_index] = sum_r_squared_val;
    shared_p_sum[local_index] = sum_p_val;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (local_index < s)
        {
            shared_r_squared[local_index] += shared_r_squared[local_index + s];
            shared_p_sum[local_index] += shared_p_sum[local_index + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0)
    {
        r_squared[blockIdx.x] = shared_r_squared[0];
        p_sum[blockIdx.x] = shared_p_sum[0];
    }
}