#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void x_calculation(float * x, float * r, float * r_squared, int size)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (index < size)
    {
        // Using register for alpha and performing computation only once per block
        float alpha = r_squared[0];
        x[index] = fmaf(alpha, r[index], x[index]); // Fused multiply-add for better performance
    }
}