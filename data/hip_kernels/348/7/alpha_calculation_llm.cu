#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void alpha_calculation(float *r_squared, float *p_sum, float *alpha)
{
    // Use blockIdx.x to support distributed computation across blocks
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check idx to ensure no out-of-bounds memory access
    if (idx == 0) {
        alpha[idx] = r_squared[idx] / p_sum[idx];
    }
}