#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void repeat_x_for_clusters(float * x, int size)
{
    // Calculate global index only once
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if the index is valid and perform the operation
    if (index < size)
    {
        x[index] = x[threadIdx.x];
    }
}