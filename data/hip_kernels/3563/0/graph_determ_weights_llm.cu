#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void graph_determ_weights(unsigned int* contact_mat_cum_row_indexes, unsigned int* contact_mat_column_indexes, 
                                     float* contact_mat_values, unsigned int rows, unsigned int values, 
                                     float* immunities, float* shedding_curve, unsigned int infection_length, 
                                     float transmission_rate, int* infection_mat_values) {

    unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimize: Initialize random state once per thread
    if (row < rows) {
        hiprandState state;
        hiprand_init(1234 + row, 0, 0, &state);
        
        unsigned int start = contact_mat_cum_row_indexes[row];
        unsigned int end = contact_mat_cum_row_indexes[row + 1];

        for (int j = start; j < end; j++) {
            float pinf_noshed = contact_mat_values[j] * transmission_rate * 
                                (1.0f - immunities[contact_mat_column_indexes[j]]);
            int delay = -1; // Default delay is -1 if no infection

            // Optimize: Use a separate flag to check exit condition
            bool infected = false;
            for (int d = 1; d <= infection_length; d++) {
                if (hiprand_uniform(&state) < pinf_noshed * shedding_curve[d - 1]) {
                    delay = d;
                    infected = true;
                    break;
                }
            }
            
            infection_mat_values[j] = delay;
        }
    }
}