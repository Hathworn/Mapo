#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void spmv_csr_vector_kernel(unsigned int computation_restriction_factor, const unsigned int* cum_row_indexes, const unsigned int* column_indexes, const float* matrix_data , const float* in_vector, float* out_vector, const unsigned int outerdim) {
    __shared__ float vals[32];
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int warp_id = thread_id / 32;
    int lane = thread_id & (32 - 1);

    int row = warp_id / computation_restriction_factor;
    if (row < outerdim) {
        int row_start = cum_row_indexes[row];
        int row_end = cum_row_indexes[row+1];
        
        // Initialize shared memory to 1 only for active lanes
        vals[threadIdx.x] = 1;
    
        for (int i = row_start + lane; i < row_end; i += 32) {
            vals[threadIdx.x] *= 1 - (matrix_data[i] * in_vector[column_indexes[i]]);
        }
        
        // Unroll reduction loops
        if (lane < 16) vals[threadIdx.x] *= vals[threadIdx.x + 16];
        if (lane < 8) vals[threadIdx.x] *= vals[threadIdx.x + 8];
        if (lane < 4) vals[threadIdx.x] *= vals[threadIdx.x + 4];
        if (lane < 2) vals[threadIdx.x] *= vals[threadIdx.x + 2];
        if (lane < 1) vals[threadIdx.x] *= vals[threadIdx.x + 1];

        // Store result from lane 0
        if (lane == 0) out_vector[row] = vals[threadIdx.x];
    }
}