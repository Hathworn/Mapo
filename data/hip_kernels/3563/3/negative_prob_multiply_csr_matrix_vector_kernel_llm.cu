#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void negative_prob_multiply_csr_matrix_vector_kernel(unsigned int* cum_row_indexes, unsigned int* column_indexes, float* matrix_data, float* in_vector, float* out_vector, unsigned int outerdim) {

    unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < outerdim) {
        float prob = 1.0;

        unsigned int row_start = cum_row_indexes[row];
        unsigned int row_end = cum_row_indexes[row+1];

        // Use shared memory for intermediate computations
        extern __shared__ float shared_probs[];
        
        unsigned int local_index = threadIdx.x;
        shared_probs[local_index] = 1.0;

        // Unroll loop for better performance
        for (int i = row_start; i < row_end; i++) {
            shared_probs[local_index] *= 1.0 - (matrix_data[i] * in_vector[column_indexes[i]]);
        }
        
        // Update global memory with computed value
        out_vector[row] = shared_probs[local_index];
    }
}