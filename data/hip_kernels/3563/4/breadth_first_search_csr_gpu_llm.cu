#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void breadth_first_search_csr_gpu(unsigned int* cum_row_indexes, unsigned int* column_indexes, int* matrix_data, unsigned int* in_infections, unsigned int* out_infections, unsigned int rows) {
    unsigned int row = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure row index is within bounds
    if (row < rows && in_infections[row] == 1) {
        // Mark this row as infected
        out_infections[row] = 1;

        unsigned int row_start = cum_row_indexes[row];
        unsigned int row_end = cum_row_indexes[row + 1];

        // Loop through the row's non-zero elements
        for (int i = row_start; i < row_end; i++) {
            int timesteps_to_transmission = matrix_data[i];
            // Check if the element can transmit infection
            if (timesteps_to_transmission > 0) {
                // Update infection status and decrement transmission step count
                if (timesteps_to_transmission == 1) {
                    out_infections[column_indexes[i]] = 1;
                }
                matrix_data[i] -= 1;
            }
        }
    }
}