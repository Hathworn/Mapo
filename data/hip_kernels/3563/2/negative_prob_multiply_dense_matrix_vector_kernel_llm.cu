#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void negative_prob_multiply_dense_matrix_vector_kernel(float* matrix, float* in_vector, float* out_vector, unsigned int outerdim, unsigned int innerdim) {
    // Optimized by pre-computing thread index and reducing divergence
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < outerdim) {
        float prob = 1.0; // Move declaration inside to avoid unnecessary computation
        unsigned int row_offset = row * innerdim; // Pre-compute row offset

        for (int i = 0; i < innerdim; i++) {
            prob *= 1.0 - (matrix[row_offset + i] * in_vector[i]);
        }
        out_vector[row] = prob; // Store result
    }
}