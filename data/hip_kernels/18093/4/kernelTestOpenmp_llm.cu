#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define CUDAMAXTHREADPERBLOCK 1024
#define CUDAMAXBLOCK 65536

using namespace std;

__global__ void kernelTestOpenmp(int *dev_b, int tt){
    // Calculate thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread operates within bounds
    if (tid < tt) {
        if (dev_b[tid] != tid) {
            printf("no!!!\n");
        }
        printf("yes!!!!\n");
    }
}