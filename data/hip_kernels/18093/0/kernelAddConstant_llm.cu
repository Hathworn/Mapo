#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define CUDAMAXTHREADPERBLOCK 1024
#define CUDAMAXBLOCK 65536

using namespace std;

__global__ void kernelAddConstant(int *g_a, const int b)
{
    // Calculate the global index for the thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if index is within bounds to prevent accessing out of bounds memory
    if (idx < CUDAMAXBLOCK * CUDAMAXTHREADPERBLOCK) {
        g_a[idx] += b;
    }
}