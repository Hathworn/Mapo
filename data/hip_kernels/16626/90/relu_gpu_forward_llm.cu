#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Calculate the global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use loop unrolling to process multiple elements per thread if applicable
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < N; i += stride) {
        // Optimize the conditional operation using max function
        out[i] = max(in[i], 0.0f);
    }
}