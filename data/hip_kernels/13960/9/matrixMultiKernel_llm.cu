#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMultiKernel(float *C, float *A, float *B, int Width) {
    const int BLOCK_SIZE = 16; // NOTE: This must be similar to line 338

    // block and thread indexes
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Dividing the matrices into sub sections
    int a_begin = Width * BLOCK_SIZE * by;
    int a_end = a_begin + Width - 1;
    int a_step = BLOCK_SIZE;

    int b_begin = BLOCK_SIZE * bx;
    int b_step = BLOCK_SIZE * Width;

    float temp_c = 0;

    // loop through the submatrices
    for (int a = a_begin, b = b_begin; a <= a_end; a += a_step, b += b_step) {
        // sub matrices
        __shared__ float sub_a[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float sub_b[BLOCK_SIZE][BLOCK_SIZE];

        // Optimize memory accesses for A and B
        int a_index = a + Width * ty + tx;
        int b_index = b + Width * ty + tx;

        sub_a[ty][tx] = A[a_index];
        sub_b[ty][tx] = B[b_index]; // Correct access pattern for B

        __syncthreads();

        // Sub matrix multiplication with pragma unroll
        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            temp_c += sub_a[ty][k] * sub_b[k][tx];
        }
        __syncthreads();
    }

    // Store the computed value in the correct output position
    int c = Width * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + Width * ty + tx] = temp_c;
}