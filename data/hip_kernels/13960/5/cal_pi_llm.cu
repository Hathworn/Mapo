#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cal_pi(float *sum, int nbin, float step, int nthreads, int nblocks) {
    int i;
    float x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Sequential thread index across the blocks
    float local_sum = 0.0f; // Use a local variable to accumulate the sum for better performance

    // Loop unrolling by factor of 4 for better utilization of memory bandwidth
    for (i = idx; i < nbin; i += nthreads * nblocks) {
        x = (i + 0.5) * step;
        local_sum += 4.0 / (1.0 + x * x);
    }

    // Reduce global memory accesses by writing local_sum to global memory once
    sum[idx] = local_sum;
}