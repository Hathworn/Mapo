#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_THREADS 20
#define pi(x) printf("%d\n",x);
#define HANDLE_ERROR(err) ( HandleError( err, __FILE__, __LINE__ ) )
#define th_p_block  256

__global__ void dotPro(long n, float *vec1, float *vec2, float *vec3) {

    // Allocate shared memory
    __shared__ float cache[th_p_block];

    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int cacheIdx = threadIdx.x;

    float temp = 0;
    // Efficient strided loop
    for (; tid < n; tid += blockDim.x * gridDim.x) {
        temp += vec1[tid] * vec2[tid];
    }

    cache[cacheIdx] = temp;
    __syncthreads();

    // Efficient reduction using loop unrolling
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (cacheIdx < stride) {
            cache[cacheIdx] += cache[cacheIdx + stride];
        }
        __syncthreads();
    }

    // Only first thread writes the result
    if (cacheIdx == 0) {
        vec3[blockIdx.x] = cache[0];
    }
}