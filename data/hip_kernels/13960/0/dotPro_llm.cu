#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_THREADS 20
#define pi(x) printf("%d\n",x);
#define HANDLE_ERROR(err) ( HandleError( err, __FILE__, __LINE__ ) )
#define th_p_block  256

__global__ void dotPro(long n, double *vec1, double *vec2, double *vec3) {

    __shared__ double cache[th_p_block];
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int cacheIdx =  threadIdx.x;
    double temp = 0;

    // Load data into shared memory and compute partial dot product
    while(tid < n) {
        temp += vec1[tid] * vec2[tid];
        tid += blockDim.x * gridDim.x;
    }
    cache[cacheIdx] = temp;
    __syncthreads();

    // Perform parallel reduction
    for(unsigned int i = blockDim.x/2; i > 0; i >>= 1) {
        if(cacheIdx < i) {
            cache[cacheIdx] += cache[cacheIdx + i];
        }
        __syncthreads(); // Ensure that all additions at this level are complete
    }

    // Write result for this block to global memory
    if(cacheIdx == 0) {
        vec3[blockIdx.x] = cache[0];
    }
}