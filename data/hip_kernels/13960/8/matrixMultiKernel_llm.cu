#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMultiKernel(double *C, double *A, double *B, int Width){
    const int BLOCK_SIZE = 16; // NOTE: This must be similar to line 338
    
    // Calculate global row and column of the matrix
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    double temp_c = 0.0;

    // Loop over the submatrices of A and B required to calculate C(row, col)
    for (int m = 0; m < Width / BLOCK_SIZE; ++m) {
        // Declaring shared memory for submatrices
        __shared__ double sub_a[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ double sub_b[BLOCK_SIZE][BLOCK_SIZE];

        // Load elements into shared memory
        sub_a[threadIdx.y][threadIdx.x] = A[row * Width + m * BLOCK_SIZE + threadIdx.x];
        sub_b[threadIdx.y][threadIdx.x] = B[(m * BLOCK_SIZE + threadIdx.y) * Width + col];

        __syncthreads();

        // Perform multiplication on the submatrices
        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            temp_c += sub_a[threadIdx.y][k] * sub_b[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Write the result to global memory
    C[row * Width + col] = temp_c;
}