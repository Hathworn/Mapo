#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cal_pi(double *sum, int nbin, double step, int nthreads, int nblocks) {
    double local_sum = 0.0;  // Use local variable to accumulate thread's contribution
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Sequential thread index across the blocks
    for (int i = idx; i < nbin; i += nthreads * nblocks) {
        double x = (i + 0.5) * step;
        local_sum += 4.0 / (1.0 + x * x);
    }
    sum[idx] = local_sum;  // Write the result to global memory at once
}