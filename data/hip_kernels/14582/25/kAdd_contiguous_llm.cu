#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate the global index for the thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    // Use strided access pattern to process multiple elements per thread
    for (; idx < numEls; idx += blockDim.x * gridDim.x) {
        dest[idx] = a[idx] + b[idx];
    }
}