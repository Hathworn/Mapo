#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void recipSummation(double* data, double* recip, int len)
{
    // Calculate the global index
    const int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Process only valid elements
    if (i < len) {
        const int j = 2 * i;
        double recip_val = recip[i]; // Cache reciprocal value
        data[j]     *= recip_val;    // Multiply and update
        data[j + 1] *= recip_val;
    }
}