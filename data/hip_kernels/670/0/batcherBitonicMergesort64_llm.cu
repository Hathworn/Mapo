#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// http://en.wikipedia.org/wiki/Bitonic_sort

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid  = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    // Optimize the nested loop structure for better performance
    for (int stage = 1; stage <= 6; ++stage) // Iterate over log2(64) stages
    {
        int stride = 1 << stage; // Calculate the stride for current stage
        for (int substage = stage; substage > 0; --substage)
        {
            int substride = 1 << substage; // Calculate the substride
            int idx = 2 * tid - (tid & (substride - 1));

            // Compare and swap elements to create bitonic sequence
            if (idx + substride < 64) {
                float a = sdata[idx];
                float b = sdata[idx + substride];
                bool dir = ((idx & stride) == 0);

                if ((a > b) == dir) { // Directional swap for sorting
                    sdata[idx] = b;
                    sdata[idx + substride] = a;
                }
            }
            __syncthreads(); // Synchronize threads after each substage
        }
    }

    d_out[tid] = sdata[tid];
}