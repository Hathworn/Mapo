#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello() {
    // Use register caching to optimize thread ID retrieval
    int tid = threadIdx.x;
    printf("Hello world! I'm thread %d\n", tid);
}