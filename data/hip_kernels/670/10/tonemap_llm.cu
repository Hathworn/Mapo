#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tonemap(float* d_x, float* d_y, float* d_log_Y, float* d_cdf_norm, float* d_r_new, float* d_g_new, float* d_b_new, float min_log_Y, float max_log_Y, float log_Y_range, int num_bins, int num_pixels_y, int num_pixels_x)
{
    int nx = num_pixels_x;
    int ny = num_pixels_y;
    int2 image_index_2d = make_int2((blockIdx.x * blockDim.x) + threadIdx.x, (blockIdx.y * blockDim.y) + threadIdx.y);

    // Optimized to prevent out of bounds calculations immediately
    if (image_index_2d.x >= nx || image_index_2d.y >= ny) return;

    int image_index_1d = (nx * image_index_2d.y) + image_index_2d.x;

    float x = d_x[image_index_1d];
    float y = d_y[image_index_1d];
    float log_Y = d_log_Y[image_index_1d];

    // Precompute values to reduce repetitive division
    float inv_log_Y_range = 1.0f / log_Y_range; 

    int bin_index = min(num_bins - 1, int((num_bins * (log_Y - min_log_Y) * inv_log_Y_range)));
    float Y_new = d_cdf_norm[bin_index];

    float inv_y = 1.0f / y;
    float X_new = x * (Y_new * inv_y);
    float Z_new = (1.0f - x - y) * (Y_new * inv_y);

    float r_new = (X_new * 3.2406f) + (Y_new * -1.5372f) + (Z_new * -0.4986f);
    float g_new = (X_new * -0.9689f) + (Y_new * 1.8758f) + (Z_new * 0.0415f);
    float b_new = (X_new * 0.0557f) + (Y_new * -0.2040f) + (Z_new * 1.0570f);

    d_r_new[image_index_1d] = r_new;
    d_g_new[image_index_1d] = g_new;
    d_b_new[image_index_1d] = b_new;
}