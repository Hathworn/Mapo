#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Calculate global thread ID
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Fetch data from global memory and calculate bin index
    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;
    
    // Perform atomic operation to update histogram
    atomicAdd(&(d_bins[myBin]), 1);
}