#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth(float * v_new, const float * v) {
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x;  // Fix thread index computation
    int numThreads = blockDim.x * gridDim.x;
    if (myIdx < numThreads) {  // Bound check to prevent out of range access
        int myLeftIdx = max(0, myIdx - 1);  // Use max to handle boundary safely
        int myRightIdx = min(numThreads - 1, myIdx + 1);  // Use min to handle boundary safely
        float myElt = v[myIdx];
        float myLeftElt = v[myLeftIdx];
        float myRightElt = v[myRightIdx];
        v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
    }
}