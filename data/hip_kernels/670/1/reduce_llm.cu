#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
    // Each thread loads its data into shared memory
    int lane = threadIdx.x % warpSize;
    s[lane] = p;

    // Warp reduction using shuffle down and a single shared memory location for results
    #pragma unroll // Unroll loop for better performance
    for (int i = warpSize/2; i > 0; i /= 2) {
        p += __shfl_down(p, i);
    }

    // Store the result of warp reduction in shared memory
    if (lane == 0) s[0] = p;

    // Synchronize to ensure the result is stored
    __syncthreads(); 
    
    return s[0];
}

__global__ void reduce(unsigned int * d_out_shared, const unsigned int * d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    if (t == 0) {
        *d_out_shared = sr;
    }
}