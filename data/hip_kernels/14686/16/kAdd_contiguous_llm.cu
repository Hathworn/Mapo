#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEls) { // Limit threads that participate to those processing within bounds
        dest[idx] = a[idx] + b[idx];
    }
}