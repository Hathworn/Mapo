#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate unique thread index
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    /* Initialize each thread with a unique seed and sequence number */
    hiprand_init(seed, tidx, 0, &state[tidx]);
}