#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Loop unrolling for memory access coalescing and reduced loop iterations
    while (eidx < numElements) {
        sum += a[eidx] * b[eidx];
        if (eidx + blockDim.x < numElements) sum += a[eidx + blockDim.x] * b[eidx + blockDim.x];
        eidx += gridDim.x * DP_BLOCKSIZE * 2;
    }
    
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Efficient parallel reduction using warp shuffle
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            shmem[threadIdx.x] += shmem[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Write result of block to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}