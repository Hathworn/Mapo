#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use threadIdx.x to ensure only one thread performs the division
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
    // Synchronization is not necessary due to single-thread operation
}