#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = __float2int_rd(labels[tx]);  // Use fast conversion
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Simplify logic for correct probability computation
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp);
            }
            correctProbs[tx] = 1.0f / __int2float_rn(numMax);  // Use fast float conversion
        }
    }
}