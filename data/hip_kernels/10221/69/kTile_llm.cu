#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate the global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total number of threads
    const int numThreads = blockDim.x * gridDim.x;

    // Use loop unrolling to reduce the number of iterations
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads * 4) {
        // Calculate x, y, and corresponding source coordinates
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Calculate target and source indices for four elements at a time
        if (i < tgtWidth * tgtHeight) {
            tgt[i] = src[srcY * srcWidth + srcX];
        }
        if (i + numThreads < tgtWidth * tgtHeight) {
            tgt[i + numThreads] = src[srcY * srcWidth + ((srcX + numThreads) % srcWidth)];
        }
        if (i + 2 * numThreads < tgtWidth * tgtHeight) {
            tgt[i + 2 * numThreads] = src[srcY * srcWidth + ((srcX + 2 * numThreads) % srcWidth)];
        }
        if (i + 3 * numThreads < tgtWidth * tgtHeight) {
            tgt[i + 3 * numThreads] = src[srcY * srcWidth + ((srcX + 3 * numThreads) % srcWidth)];
        }
    }
}