#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    
    // Initialize shared memory and compute partial results
    float sum = 0.0f;
    for (int i = idx; i < dim; i += blockDim.x) {
        sum += dY[i] * Y[i];
    }
    reduction_buffer[idx] = sum;
    __syncthreads();

    // Reduction phase: compute total sum using parallel reduction
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (idx < offset) {
            reduction_buffer[idx] += reduction_buffer[idx + offset];
        }
        __syncthreads();
    }

    // Fetch final result from reduction
    float total_sum = reduction_buffer[0];
    
    // Compute gradient
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - total_sum);
    }
}