#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_disp_veloc_kernel(float * displ, float * veloc, float * accel, const int size, const float deltat, const float deltatsqover2, const float deltatover2) {
    // Calculate global thread ID in a 1D grid configuration
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Ensure thread operates only within bounds
    if (id < size) {
        // Update displacement using current velocity and acceleration
        displ[id] += deltat * veloc[id] + deltatsqover2 * accel[id];
        // Update velocity using current acceleration
        veloc[id] += deltatover2 * accel[id];
        // Reset acceleration to zero
        accel[id] = 0.0f;
    }
}