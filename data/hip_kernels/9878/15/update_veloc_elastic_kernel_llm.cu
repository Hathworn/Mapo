#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel using block and thread ID calculations
__global__ void update_veloc_elastic_kernel(float *veloc, const float *accel, const int size, const float deltatover2) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure id is within bounds
    if (id < size) {
        int index1 = id;
        int index2 = size + id;
        int index3 = 2 * size + id;
        
        // Optimized memory accesses with pre-computed indices
        float accel1 = accel[index1];
        float accel2 = accel[index2];
        float accel3 = accel[index3];

        veloc[index1] += deltatover2 * accel1;
        veloc[index2] += deltatover2 * accel2;
        veloc[index3] += deltatover2 * accel3;
    }
}