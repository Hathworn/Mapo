#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prepare_boundary_potential_on_device(const float * d_potential_dot_dot_acoustic, float * d_send_potential_dot_dot_buffer, const int num_interfaces, const int max_nibool_interfaces, const int * d_nibool_interfaces, const int * d_ibool_interfaces){
    int id = threadIdx.x + blockIdx.x * blockDim.x + gridDim.x * blockDim.x * (threadIdx.y + blockIdx.y * blockDim.y);

    // Iterate over interfaces
    for (int iinterface = 0; iinterface < num_interfaces; iinterface++) {
        // Check if thread ID is within the number of elements for the current interface
        if (id < d_nibool_interfaces[iinterface]) {
            int iloc = id + max_nibool_interfaces * iinterface;
            int iglob = d_ibool_interfaces[iloc] - 1;

            // Copy potential data from global to interface buffer
            d_send_potential_dot_dot_buffer[iloc] = d_potential_dot_dot_acoustic[iglob];
        }
    }
}