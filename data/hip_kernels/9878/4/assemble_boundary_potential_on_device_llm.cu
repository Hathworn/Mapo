#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void assemble_boundary_potential_on_device(float * d_potential_dot_dot_acoustic, const float * d_send_potential_dot_dot_buffer, const int num_interfaces, const int max_nibool_interfaces, const int * d_nibool_interfaces, const int * d_ibool_interfaces) {
    // Calculate unique thread ID
    int id = threadIdx.x + blockIdx.x * blockDim.x + gridDim.x * blockDim.x * (threadIdx.y + blockIdx.y * blockDim.y);
    
    // Loop over interfaces
    for (int iinterface = 0; iinterface < num_interfaces; iinterface++) {
        if (id < d_nibool_interfaces[iinterface]) {
            // Calculate local and global indices
            int iloc = id + max_nibool_interfaces * iinterface;
            int iglob = d_ibool_interfaces[iloc] - 1;
            // Safely add to global memory using atomic operation
            atomicAdd(&d_potential_dot_dot_acoustic[iglob], d_send_potential_dot_dot_buffer[iloc]);
        }
    }
}