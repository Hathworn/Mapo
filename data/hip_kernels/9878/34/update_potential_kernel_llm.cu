#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_potential_kernel(float * potential_acoustic, float * potential_dot_acoustic, float * potential_dot_dot_acoustic, const int size, const float deltat, const float deltatsqover2, const float deltatover2) {
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    if (id < size) {
        // Update potential values
        float dot_acoustic = potential_dot_acoustic[id];
        float dot_dot_acoustic = potential_dot_dot_acoustic[id];
        
        potential_acoustic[id] += deltat * dot_acoustic + deltatsqover2 * dot_dot_acoustic;
        potential_dot_acoustic[id] += deltatover2 * dot_dot_acoustic;
        potential_dot_dot_acoustic[id] = 0.0f;
    }
}