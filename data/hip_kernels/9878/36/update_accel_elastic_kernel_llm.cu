#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_accel_elastic_kernel(float * accel, const float * veloc, const int size, const float two_omega_earth, const float * rmassx, const float * rmassy, const float * rmassz){
    // Calculate unique global thread ID
    int id = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * gridDim.x * blockDim.x;

    if (id < size) {
        // Precompute index base for the current thread to reduce redundant calculations
        int idx3 = id * 3;

        // Update the acceleration with elimination of redundant calculations
        float vel_x = veloc[idx3];
        float vel_y = veloc[idx3 + 1];
        accel[idx3]     = accel[idx3] * rmassx[id] + two_omega_earth * vel_y;
        accel[idx3 + 1] = accel[idx3 + 1] * rmassy[id] - two_omega_earth * vel_x;
        accel[idx3 + 2] = accel[idx3 + 2] * rmassz[id];
    }
}