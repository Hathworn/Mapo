#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_veloc_acoustic_kernel(float *veloc, const float *accel, const int size, const float deltatover2) {
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x; // Simplified id calculation

    if (id < size) {
        veloc[id] += deltatover2 * accel[id]; // Simplified velocity update
    }
}