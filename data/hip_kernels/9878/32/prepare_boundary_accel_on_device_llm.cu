#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prepare_boundary_accel_on_device(const float *d_accel, float *d_send_accel_buffer, const int num_interfaces, const int max_nibool_interfaces, const int *d_nibool_interfaces, const int *d_ibool_interfaces) {
    int id = threadIdx.x + blockIdx.x * blockDim.x + gridDim.x * blockDim.x * (threadIdx.y + blockIdx.y * blockDim.y);
  
    // Using shared memory to avoid thread divergence in loop
    __shared__ int shared_interfaces[32]; // Assume max interfaces are 32, adjust if needed
    
    for (int iinterface = threadIdx.x; iinterface < num_interfaces; iinterface += blockDim.x) {
        shared_interfaces[iinterface] = d_nibool_interfaces[iinterface];
    }
    __syncthreads();

    for (int iinterface = 0; iinterface < num_interfaces; iinterface++) {
        if (id < shared_interfaces[iinterface]) {
            int iloc = id + max_nibool_interfaces * iinterface;
            int iglob = d_ibool_interfaces[iloc] - 1;
            
            // Optimize memory access pattern
            int buffer_index = iloc * 3;
            int accel_index = iglob * 3;
            d_send_accel_buffer[buffer_index] = d_accel[accel_index];
            d_send_accel_buffer[buffer_index + 1] = d_accel[accel_index + 1];
            d_send_accel_buffer[buffer_index + 2] = d_accel[accel_index + 2];
        }
    }
}