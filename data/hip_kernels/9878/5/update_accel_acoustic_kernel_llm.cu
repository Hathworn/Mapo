#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_accel_acoustic_kernel(float *accel, const int size, const float *rmass) {
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Ensure this thread processes within bounds
    if (id < size) {
        // Update acceleration with reciprocal mass
        accel[id] *= rmass[id];
    }
}