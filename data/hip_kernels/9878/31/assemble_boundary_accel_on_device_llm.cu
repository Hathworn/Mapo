#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void assemble_boundary_accel_on_device(float * d_accel, const float * d_send_accel_buffer, const int num_interfaces, const int max_nibool_interfaces, const int * d_nibool_interfaces, const int * d_ibool_interfaces) {
    int id = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * blockIdx.y) + blockDim.x * blockDim.y * blockDim.z * blockIdx.x; // Simplified global thread ID calculation
    if(id >= max_nibool_interfaces * num_interfaces) return; // Boundary check

    int iinterface = id / max_nibool_interfaces; // Calculate interface index
    int local_id = id % max_nibool_interfaces; // Calculate local index in the interface

    if (local_id < d_nibool_interfaces[iinterface]) {
        int iloc = local_id + max_nibool_interfaces * iinterface;
        int iglob = d_ibool_interfaces[iloc] - 1;

        // Use direct indexing for atomic additions
        atomicAdd(&d_accel[iglob * 3 + 0], d_send_accel_buffer[iloc * 3 + 0]);
        atomicAdd(&d_accel[iglob * 3 + 1], d_send_accel_buffer[iloc * 3 + 1]);
        atomicAdd(&d_accel[iglob * 3 + 2], d_send_accel_buffer[iloc * 3 + 2]);
    }
}