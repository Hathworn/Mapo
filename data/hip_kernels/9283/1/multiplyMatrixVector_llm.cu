#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Prototype matrix representation. */
struct dag_array_t {
    size_t rows;
    size_t cols;
    int* matrix;
};

/*
DAG Primitive. 
*/
const int tp_TILE_DIM = 32;
const int tp_BLOCK_ROWS = 8;

// Optimized global kernel function
__global__ void multiplyMatrixVector(int* result, int* matrix, int* vector, int cols) {
    __shared__ int reduce_array[tp_TILE_DIM]; // Use tp_TILE_DIM for shared memory size

    int vector_slice_offset = blockIdx.x * cols + threadIdx.x;
    int matrix_slice_offset = blockIdx.y * cols + threadIdx.x;

    // Load data into shared memory
    reduce_array[threadIdx.x] = matrix[matrix_slice_offset] * vector[vector_slice_offset];

    __syncthreads();

    // Parallel reduction
    for (int stride = tp_BLOCK_ROWS / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            reduce_array[threadIdx.x] += reduce_array[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write result
    if (threadIdx.x == 0) {
        result[blockIdx.x * cols + blockIdx.y] = reduce_array[0];
    }
}