#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Prototype matrix representation. */
struct dag_array_t{
    size_t rows;
    size_t cols;
    int* matrix;
};

/* Constants for shared memory tiling. */
const int tp_TILE_DIM = 32;
const int tp_BLOCK_ROWS = 8;

__global__ void transposeCoalesced(int *result, const int *in)
{
    const int TILE_DIM = tp_TILE_DIM;
    const int BLOCK_ROWS = tp_BLOCK_ROWS;

    __shared__ int tile[TILE_DIM][TILE_DIM + 1];  // +1 to avoid bank conflicts

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    // Load data into shared memory
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[threadIdx.y + j][threadIdx.x] = in[(y + j) * width + x];

    __syncthreads();

    // Calculate transposed coordinates
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    // Write transposed data
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        result[(y + j) * width + x] = tile[threadIdx.x][threadIdx.y + j];
}