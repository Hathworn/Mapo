#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_DIM_small 64
#define BLOCK_DIM 256

static const unsigned int threads = 512;
bool print_intermediate_arrays = false;
const char *BASE_PATH = "/home/giuliano/git/cuda/reduction";

const char *kern_0 = "filter_roi";
const char *kern_1 = "imperviousness_change_histc_sh_4";
const char *kern_2 = "imperviousness_change";
char buffer[255];

const char *FIL_ROI = "/media/DATI/db-backup/ssgci-data/testing/ssgci_roi.tif";
const char *FIL_BIN1 = "/media/DATI/db-backup/ssgci-data/testing/ssgci_bin.tif";
const char *FIL_BIN2 = "/media/DATI/db-backup/ssgci-data/testing/ssgci_bin2.tif";

const char *FIL_LTAKE_grid = "/home/giuliano/git/cuda/reduction/data/LTAKE_map.tif";
const char *FIL_LTAKE_count = "/home/giuliano/git/cuda/reduction/data/LTAKE_count.txt";

__global__ void imperviousness_change_char(const unsigned char *dev_BIN1, const unsigned char *dev_BIN2, unsigned int WIDTH, unsigned int HEIGHT, char *dev_LTAKE_map)
{
    unsigned long int tix = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    // Optimization: Checking boundary within the main loop
    if (tix < WIDTH * HEIGHT) {
        dev_LTAKE_map[tix] = dev_BIN2[tix] - dev_BIN1[tix]; // Compute difference and store
    }
}