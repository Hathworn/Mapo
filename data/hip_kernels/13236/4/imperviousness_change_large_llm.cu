#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_DIM_small 64
#define BLOCK_DIM 256

static const unsigned int threads = 512;
bool print_intermediate_arrays = false;
const char *BASE_PATH = "/home/giuliano/git/cuda/reduction";

const char *kern_0 = "filter_roi";
const char *kern_1 = "imperviousness_change_histc_sh_4";
const char *kern_2 = "imperviousness_change";
char buffer[255];

const char *FIL_ROI = "/media/DATI/db-backup/ssgci-data/testing/ssgci_roi.tif";
const char *FIL_BIN1 = "/media/DATI/db-backup/ssgci-data/testing/ssgci_bin.tif";
const char *FIL_BIN2 = "/media/DATI/db-backup/ssgci-data/testing/ssgci_bin2.tif";

const char *FIL_LTAKE_grid = "/home/giuliano/git/cuda/reduction/data/LTAKE_map.tif";
const char *FIL_LTAKE_count = "/home/giuliano/git/cuda/reduction/data/LTAKE_count.txt";

/* Kernel optimized */
__global__ void imperviousness_change_large(const unsigned char *dev_BIN1, const unsigned char *dev_BIN2, unsigned int WIDTH, unsigned int HEIGHT, int *dev_LTAKE_map, int mapel_per_thread) {
    unsigned long int tid = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned long int tix = tid * mapel_per_thread;

    if (tix < WIDTH * HEIGHT) { // Avoid unnecessary computations
        for (long int ii = 0; ii < mapel_per_thread; ii++) {
            if (tix + ii < WIDTH * HEIGHT) {
                dev_LTAKE_map[tix + ii] = static_cast<int>(dev_BIN2[tix + ii]) - static_cast<int>(dev_BIN1[tix + ii]);
            }
        }
    }
}