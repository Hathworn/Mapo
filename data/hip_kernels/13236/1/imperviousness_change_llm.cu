#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_DIM_small 64
#define BLOCK_DIM 256

static const unsigned int threads = 512;
bool print_intermediate_arrays = false;
const char *BASE_PATH = "/home/giuliano/git/cuda/reduction";

const char *kern_0 = "filter_roi";
const char *kern_1 = "imperviousness_change_histc_sh_4";
const char *kern_2 = "imperviousness_change";
char buffer[255];

const char *FIL_ROI = "/media/DATI/db-backup/ssgci-data/testing/ssgci_roi.tif";
const char *FIL_BIN1 = "/media/DATI/db-backup/ssgci-data/testing/ssgci_bin.tif";
const char *FIL_BIN2 = "/media/DATI/db-backup/ssgci-data/testing/ssgci_bin2.tif";

const char *FIL_LTAKE_grid = "/home/giuliano/git/cuda/reduction/data/LTAKE_map.tif";
const char *FIL_LTAKE_count = "/home/giuliano/git/cuda/reduction/data/LTAKE_count.txt";

__global__ void imperviousness_change(const unsigned char *dev_BIN1, const unsigned char *dev_BIN2, unsigned int WIDTH, unsigned int HEIGHT, int *dev_LTAKE_map) {
    // Calculate the global thread index
    unsigned long int tix = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread index is within bounds
    if (tix < WIDTH * HEIGHT) {
        // Compute the difference and store the result
        dev_LTAKE_map[tix] = static_cast<int>(dev_BIN2[tix]) - static_cast<int>(dev_BIN1[tix]);
    }
}