#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_DIM_small 64
#define BLOCK_DIM 256

static const unsigned int threads = 512;
bool print_intermediate_arrays = false;
const char *BASE_PATH = "/home/giuliano/git/cuda/reduction";

const char *kern_0 = "filter_roi";
const char *kern_1 = "imperviousness_change_histc_sh_4";
const char *kern_2 = "imperviousness_change";
char buffer[255];

const char *FIL_ROI = "/media/DATI/db-backup/ssgci-data/testing/ssgci_roi.tif";
const char *FIL_BIN1 = "/media/DATI/db-backup/ssgci-data/testing/ssgci_bin.tif";
const char *FIL_BIN2 = "/media/DATI/db-backup/ssgci-data/testing/ssgci_bin2.tif";

const char *FIL_LTAKE_grid = "/home/giuliano/git/cuda/reduction/data/LTAKE_map.tif";
const char *FIL_LTAKE_count = "/home/giuliano/git/cuda/reduction/data/LTAKE_count.txt";

__global__ void filter_roi(unsigned char *BIN, const unsigned char *ROI, unsigned int map_len) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int gridSize = blockDim.x * gridDim.x;

    // Loop with grid-stride to cover entire data range
    for (; i < map_len; i += gridSize) {
        BIN[i] = static_cast<unsigned char>(static_cast<int>(BIN[i]) * static_cast<int>(ROI[i]));
    }
}