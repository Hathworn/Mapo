#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DiamondStep(uint8_t* matrix, unsigned *random, int currentSize, int matrixSize, int randValue)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int half = currentSize / 2;
    int minRand = -randValue;
    int row = y * currentSize + half;
    int col = x * currentSize + half;

    // Check boundaries to avoid unnecessary computations
    if (row < half || row >= matrixSize - half || col < half || col >= matrixSize - half) return;

    // Cache values for improved memory access
    int topLeft = matrix[(row - half) * matrixSize + (col - half)];
    int topRight = matrix[(row - half) * matrixSize + (col + half)];
    int bottomLeft = matrix[(row + half) * matrixSize + (col - half)];
    int bottomRight = matrix[(row + half) * matrixSize + (col + half)];
    int randomValue = random[row * matrixSize + col] % (randValue - minRand) + minRand;

    // Calculate the value
    int value = (topLeft + topRight + bottomLeft + bottomRight + randomValue) / 4;
    matrix[row * matrixSize + col] = value;
}