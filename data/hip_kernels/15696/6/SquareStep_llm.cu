#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SquareStep(uint8_t* matrix, unsigned* random, int currentSize, int matrixSize, int maxRowThread, int maxColThread, int randValue)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    // Check bounds to avoid memory access outside the matrix
    if (x < maxRowThread && y < maxColThread)
    {
        int half = currentSize / 2;
        int minRand = -randValue;
        int div = 0;
        int value = 0;

        // Calculate element positions based on x and y and currentSize
        int elemX = x * currentSize * (y % 2 == 0) + y * half * (y % 2 != 0);
        int elemY = (y * half + half) * (y % 2 == 0) + x * currentSize * (y % 2 != 0);

        // Use conditions to avoid divergent branches and to ensure correct index checks
        bool condXMin = elemX > 0;
        value += condXMin ? matrix[(elemX - half) * matrixSize + elemY] : 0;
        div += condXMin;

        bool condXMax = elemX < (matrixSize - 1);
        value += condXMax ? matrix[(elemX + half) * matrixSize + elemY] : 0;
        div += condXMax;

        bool condYMin = elemY > 0;
        value += condYMin ? matrix[elemX * matrixSize + elemY - half] : 0;
        div += condYMin;

        bool condYMax = elemY < (matrixSize - 1);
        value += condYMax ? matrix[elemX * matrixSize + elemY + half] : 0;
        div += condYMax;

        // Update matrix with computed value
        value += (minRand + random[elemX * matrixSize + elemY] % (randValue - minRand));
        matrix[elemX * matrixSize + elemY] = value / div;
    }
}