#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_one_vec(float* d_one_vec, size_t length)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use loop to cover entire array with fewer threads
    for (; i < length; i += stride) {
        d_one_vec[i] = 1.f; // Set the vector element to 1
    }
}