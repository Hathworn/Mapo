#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_loss_kernel(float *reduced_loss, float *predict, float *target, float *workspace, int batch_size, int num_outputs)
{
    int batch_idx = blockDim.x * blockIdx.x + threadIdx.x;
    extern __shared__ float s_data[];
    float loss = 0.f;

    // Each thread calculates entropy for each data and accumulates to shared memory
    if(batch_idx < batch_size) {
        for (int c = 0; c < num_outputs; c++) {
            // Use of faster memory access techniques
            int idx = batch_idx * num_outputs + c;
            loss += target[idx] * logf(predict[idx]);
        }
        workspace[batch_idx] = -loss;
    } else {
        workspace[batch_idx] = 0.0f;
    }

    // Reduction using a single thread block if blockIdx.x == 0
    if (blockIdx.x > 0) return;

    s_data[threadIdx.x] = 0.f;

    // Cumulate workspace data with boundary check
    for (int i = threadIdx.x; i < batch_size; i += blockDim.x) {
        s_data[threadIdx.x] += workspace[i];
    }

    __syncthreads();

    // Reduction using warp shuffle for performance improvement
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride && threadIdx.x + stride < batch_size) {
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write the final result to the output array
    if (threadIdx.x == 0) {
        reduced_loss[blockIdx.x] = s_data[0];
    }
}