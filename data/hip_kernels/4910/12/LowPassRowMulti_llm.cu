#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LowPassRowMulti(float *d_Result, float *d_Data, int width, int pitch, int height) {
    __shared__ float data[CONVROW_W + 2 * RADIUS];
    const int tx = threadIdx.x;
    const int block = blockIdx.x / (NUM_SCALES + 3);
    const int scale = blockIdx.x % (NUM_SCALES + 3);
    const int xout = block * CONVROW_W + tx;
    const int loadPos = xout - RADIUS;
    const int yptr = blockIdx.y * pitch;
    const int writePos = yptr + height * pitch * scale + xout;
    float *kernel = d_Kernel + scale * 16;

    // Better boundary checking with min/max
    if (loadPos < 0) {
        data[tx] = d_Data[yptr];
    } else if (loadPos >= width) {
        data[tx] = d_Data[yptr + width - 1];
    } else {
        data[tx] = d_Data[yptr + loadPos];
    }
    __syncthreads();

    // Avoid redundant computations and use loop for unrolling operation
    if (xout < width && tx < CONVROW_W) {
        float sum = 0.0f;
        for (int i = 0; i < 5; ++i) {
            sum += (data[tx + i] + data[tx + 8 - i]) * kernel[i];
        }
        d_Result[writePos] = sum;
    }
    __syncthreads();
}