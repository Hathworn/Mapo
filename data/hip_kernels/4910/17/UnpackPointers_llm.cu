#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    // Load common variables outside the loops for efficiency
    const int tx = threadIdx.x;
    int numPts = 0;
    const int stride = w / 32;
    // Use shared memory to reduce global memory access
    __shared__ unsigned int shared_minmax[32];

    for (int y = 0; y < h / 32; y++) {
        for (int x = 0; x < w; x += 32) {
            // Load data to shared memory to reduce global memory access
            shared_minmax[tx] = minmax[y * stride + x + tx];
            __syncthreads();

            unsigned int val = shared_minmax[tx];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[32 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
            __syncthreads();
        }
    }
}