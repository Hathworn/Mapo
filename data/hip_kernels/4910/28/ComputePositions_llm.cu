#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputePositions(float *g_Data1, float *g_Data2, float *g_Data3, int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index computation
    if (i >= numPts) return;

    int p = d_Ptrs[i];

    // Load values from global memory into registers for faster access
    float val0 = g_Data2[p];
    float val1 = g_Data2[p-1];
    float val2 = g_Data2[p+1];
    float val3 = g_Data2[p-w];
    float val4 = g_Data2[p+w];
    float val5 = g_Data3[p];
    float val6 = g_Data1[p];

    // Precompute reusable terms to reduce operations
    float dx = 0.5f * (val2 - val1);
    float dxx = 2.0f * val0 - val1 - val2;
    float dy = 0.5f * (val4 - val3);
    float dyy = 2.0f * val0 - val3 - val4;
    float ds = 0.5f * (val6 - val5);
    float dss = 2.0f * val0 - val5 - val6;

    float dxy = 0.25f * (g_Data2[p+w+1] + g_Data2[p-w-1] - g_Data2[p-w+1] - g_Data2[p+w-1]);
    float dxs = 0.25f * (g_Data3[p+1] + g_Data1[p-1] - g_Data1[p+1] - g_Data3[p-1]);
    float dys = 0.25f * (g_Data3[p+w] + g_Data1[p-w] - g_Data3[p-w] - g_Data1[p+w]);

    float idxx = dyy * dss - dys * dys;
    float idxy = dys * dxs - dxy * dss;
    float idxs = dxy * dys - dyy * dxs;
    float idyy = dxx * dss - dxs * dxs;
    float idys = dxy * dxs - dxx * dys;
    float idss = dxx * dyy - dxy * dxy;

    // Use one division and multiply by reciprocal for subsequent operations to increase efficiency
    float det = idxx * dxx + idxy * dxy + idxs * dxs;
    float idet = 1.0f / det;

    float pdx = idet * (idxx * dx + idxy * dy + idxs * ds);
    float pdy = idet * (idxy * dx + idyy * dy + idys * ds);
    float pds = idet * (idxs * dx + idys * dy + idss * ds);

    // Remove branching by calculating clamped values in all cases
    float clamped_pdx = (pdx < -0.5f || pdx > 0.5f) ? __fdividef(dx, dxx) : pdx;
    float clamped_pdy = (pdy < -0.5f || pdy > 0.5f) ? __fdividef(dy, dyy) : pdy;
    float clamped_pds = (pds < -0.5f || pds > 0.5f) ? __fdividef(ds, dss) : pds;

    float dval = 0.5f * (dx * clamped_pdx + dy * clamped_pdy + ds * clamped_pds);
    d_Sift[i + 0 * maxPts] = (p % w) + clamped_pdx;
    d_Sift[i + 1 * maxPts] = (p / w) + clamped_pdy;
    d_Sift[i + 2 * maxPts] = d_ConstantA[0] * exp2f(clamped_pds * d_ConstantB[0]);
    d_Sift[i + 3 * maxPts] = val0 + dval;

    float tra = dxx + dyy;
    det = dxx * dyy - dxy * dxy;
    d_Sift[i + 4 * maxPts] = __fdividef(tra * tra, det);
}