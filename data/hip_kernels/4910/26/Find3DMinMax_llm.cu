#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int height)
{
    //Data cache
    __shared__ float data1[3 * (MINMAX_SIZE + 2)];
    __shared__ float data2[3 * (MINMAX_SIZE + 2)];
    __shared__ float data3[3 * (MINMAX_SIZE + 2)];
    __shared__ float ymin1[(MINMAX_SIZE + 2)];
    __shared__ float ymin2[(MINMAX_SIZE + 2)];
    __shared__ float ymin3[(MINMAX_SIZE + 2)];
    __shared__ float ymax1[(MINMAX_SIZE + 2)];
    __shared__ float ymax2[(MINMAX_SIZE + 2)];
    __shared__ float ymax3[(MINMAX_SIZE + 2)];

    // Current tile and apron limits, relative to row start
    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_SIZE;  // Changed __mul24 to *
    const int xEnd = xStart + MINMAX_SIZE - 1;
    const int xReadPos = xStart + tx - WARP_SIZE;
    const int xWritePos = xStart + tx;
    const int xEndClamped = min(xEnd, width - 1);
    int memWid = MINMAX_SIZE + 2;

    int memPos0 = (tx - WARP_SIZE + 1);
    int memPos1 = (tx - WARP_SIZE + 1);
    int yq = 0;
    unsigned int output = 0;
    for (int y = 0; y < 34; y++) {
        output >>= 1;
        int memPos = yq * memWid + (tx - WARP_SIZE + 1);
        int yp = 32 * blockIdx.y + y - 1;
        yp = max(yp, 0);
        yp = min(yp, height - 1);
        int readStart = yp * width;  // Changed __mul24 to *

        // Set the entire data cache contents
        if (tx >= (WARP_SIZE - 1)) {
            // Use unified assignments to avoid repetitive conditions
            if (xReadPos < 0 || xReadPos >= width) {
                data1[memPos] = 0;
                data2[memPos] = 0;
                data3[memPos] = 0;
            } else {
                data1[memPos] = d_Data1[readStart + xReadPos];
                data2[memPos] = d_Data2[readStart + xReadPos];
                data3[memPos] = d_Data3[readStart + xReadPos];
            }
        }
        __syncthreads();

        int memPos2 = yq * memWid + tx;
        if (y > 1) {
            if (tx < memWid) {
                // Use temporary variable for repetitive calculations
                float d1_0 = data1[memPos0], d1_1 = data1[memPos1], d1_2 = data1[memPos2];
                float d2_0 = data2[memPos0], d2_1 = data2[memPos1], d2_2 = data2[memPos2];
                float d3_0 = data3[memPos0], d3_1 = data3[memPos1], d3_2 = data3[memPos2];

                float min1 = fminf(fminf(d1_0, d1_1), d1_2);
                float min2 = fminf(fminf(d2_0, d2_1), d2_2);
                float min3 = fminf(fminf(d3_0, d3_1), d3_2);

                float max1 = fmaxf(fmaxf(d1_0, d1_1), d1_2);
                float max2 = fmaxf(fmaxf(d2_0, d2_1), d2_2);
                float max3 = fmaxf(fmaxf(d3_0, d3_1), d3_2);

                ymin1[tx] = min1;
                ymin2[tx] = fminf(fminf(min1, min2), min3);
                ymin3[tx] = min3;

                ymax1[tx] = max1;
                ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
                ymax3[tx] = max3;
            }
        }
        __syncthreads();

        if (y > 1) {
            if (tx < MINMAX_SIZE) {
                if (xWritePos <= xEndClamped) {
                    // Reduce nested fminf and fmaxf by using temporary variables
                    float min_val = fminf(fminf(ymin2[tx], ymin2[tx + 2]),
                                      fminf(ymin1[tx + 1], ymin3[tx + 1]));
                    min_val = fminf(min_val, fminf(data2[memPos0 + 1], data2[memPos2 + 1]));
                    min_val = fminf(min_val, d_ConstantA[1]);

                    float max_val = fmaxf(fmaxf(ymax2[tx], ymax2[tx + 2]),
                                      fmaxf(ymax1[tx + 1], ymax3[tx + 1]));
                    max_val = fmaxf(max_val, fmaxf(data2[memPos0 + 1], data2[memPos2 + 1]));
                    max_val = fmaxf(max_val, d_ConstantA[0]);

                    if (data2[memPos1 + 1] < min_val || data2[memPos1 + 1] > max_val)
                        output |= 0x80000000;
                }
            }
        }
        __syncthreads();

        memPos0 = memPos1;
        memPos1 = memPos2;
        yq = (yq < 2 ? yq + 1 : 0);
    }
    if (tx < MINMAX_SIZE && xWritePos < width) {
        int writeStart = blockIdx.y * width + xWritePos;  // Changed __mul24 to *
        d_Result[writeStart] = output;
    }
}