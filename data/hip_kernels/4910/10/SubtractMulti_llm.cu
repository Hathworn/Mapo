#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Simplify thread indexing
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Simplify thread indexing
    int sz = height * pitch;
    int p = threadIdx.z * sz + y * pitch + x;

    if (x < width && y < height) {
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
    // __syncthreads() is unnecessary outside of shared memory usage
}