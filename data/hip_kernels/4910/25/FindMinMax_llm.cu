#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized FindMinMax kernel
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    int p = y * width + x;
    
    float val = (x < width) ? d_Data[p] : d_Data[p - x];
    minvals[tx] = val;
    maxvals[tx] = val;

    // Loop unrolling for better performance
    #pragma unroll
    for (int ty = 1; ty < 16; ++ty) {
        p += width;
        if (x < width) {
            val = d_Data[p];
            minvals[tx] = fminf(minvals[tx], val); // Use intrinsic for min
            maxvals[tx] = fmaxf(maxvals[tx], val); // Use intrinsic for max
        }
    }
    __syncthreads();

    // Reduction with loop unrolling
    for (int offset = b >> 1; offset > 0; offset >>= 1) {
        if (tx < offset) {
            minvals[tx] = fminf(minvals[tx], minvals[tx + offset]);
            maxvals[tx] = fmaxf(maxvals[tx], maxvals[tx + offset]);
        }
        __syncthreads();
    }
    
    // Write results
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr + 0] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}