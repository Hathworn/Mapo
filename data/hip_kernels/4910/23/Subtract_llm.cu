#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Calculate thread's unique index using 2D grid and block dimensions
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Compute 1D array index
    int p = y * width + x;
    
    // Perform subtraction if within bounds
    if (x < width && y < height)
        d_Result[p] = d_Data1[p] - d_Data2[p];
}