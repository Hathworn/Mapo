#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;  // Use actual blockDim.x for better flexibility
    const int y = blockIdx.y * blockDim.y + threadIdx.y;  // Use actual blockDim.y for better flexibility
    int p = y * pitch + x;

    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // __syncthreads() is not necessary here as there is no inter-thread dependency
}