#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Calculate global thread index in the x and y directions.
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within bounds before computing the index.
    if (x < width && y < height) {
        int p = y * width + x; // Simplify the global index calculation.
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
    // __syncthreads() is removed as it was unnecessary here since threads do not need synchronization.
}