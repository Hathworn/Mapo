#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(long* a, long* b, long* c, long N) {
    long baseIdx = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index
    if (baseIdx < N) { // Ensure index is within bounds
        c[baseIdx] = a[baseIdx] * b[baseIdx];
    }
    __syncthreads();

    for (long step = blockDim.x / 2; step > 0; step /= 2) { // Use loop unrolling for reduction
        if (baseIdx < N && baseIdx < step) {
            c[baseIdx] += c[baseIdx + step];
        }
        __syncthreads();
    }
}