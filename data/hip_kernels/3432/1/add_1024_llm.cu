#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_1024(long* a, long* b, long* c, long N) {
    // Calculate global index
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Perform element-wise multiplication
    if (idx < N) {
        c[idx] = a[idx] * b[idx];
    }
    __syncthreads();

    // Reduction using shared memory for better performance
    extern __shared__ long shared_c[];
    if (idx < N) {
        shared_c[threadIdx.x] = c[idx];
    }
    __syncthreads();

    for (long step = blockDim.x / 2; step > 0; step /= 2) {
        if (threadIdx.x < step && threadIdx.x + step < N) {
            shared_c[threadIdx.x] += shared_c[threadIdx.x + step];
        }
        __syncthreads();
    }

    // Write result back to global memory
    if (threadIdx.x == 0) {
        c[blockIdx.x] = shared_c[0];
    }
}