#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_sqrt(float *dist, int width, int pitch, int k) {
    // Use shared memory loading
    __shared__ float sharedMem[32];  // Adjust size based on blockDim

    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIndex < width && yIndex < k) {
        // Load data into shared memory
        sharedMem[threadIdx.x] = dist[yIndex * pitch + xIndex];

        __syncthreads();  // Synchronize threads

        // Compute sqrt using shared memory
        float val = sqrt(sharedMem[threadIdx.x]);

        // Write back to global memory
        dist[yIndex * pitch + xIndex] = val;
    }
}