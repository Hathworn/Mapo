#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_distance_texture(hipTextureObject_t ref, int ref_width, float* query, int query_width, int query_pitch, int height, float* dist) {
    // Calculate global thread index
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIndex < query_width && yIndex < ref_width) {
        float ssd = 0.f;

        // Unroll the loop for efficiency
        #pragma unroll
        for (int i = 0; i < height; i++) {
            float tmp = tex2D<float>(ref, (float)yIndex, (float)i) - query[i * query_pitch + xIndex];
            ssd += tmp * tmp;
        }

        // Write the result to global memory
        dist[yIndex * query_pitch + xIndex] = ssd;
    }
}