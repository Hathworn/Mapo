#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void add_query_points_norm_and_sqrt(float *array, int width, int pitch, int k, float *norm) {
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIndex < width && yIndex < k) {
        int index = yIndex * pitch + xIndex;  // Calculate index once for reuse
        float value = array[index] + norm[xIndex];  // Pre-calculate repeated expressions
        array[index] = sqrt(value);  // Assign square root of the calculated value
    }
}