#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_squared_norm(float * array, int width, int pitch, int height, float * norm){
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (xIndex < width) {
        float sum = 0.0f;
        int y = threadIdx.y;

        // Use shared memory for data fetched within the block
        __shared__ float sharedData[1024]; 

        // Process rows in parallel
        for (int i = y; i < height; i += blockDim.y) {
            float val = array[i * pitch + xIndex];
            sharedData[y] = val * val;
            __syncthreads();

            if (y == 0) {
                for (int j = 0; j < blockDim.y && (i+j) < height; ++j) {
                    sum += sharedData[j];
                }
            }
        }

        // Store result to global memory, single thread update to avoid race condition
        if (y == 0) {
            norm[xIndex] = sum;
        }
    }
}