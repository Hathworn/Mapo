#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_reference_points_norm(float * array, int width, int pitch, int height, float * norm) {
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int xIndex = blockIdx.x * blockDim.x + tx;
    unsigned int yIndex = blockIdx.y * blockDim.y + ty;

    __shared__ float shared_vec[16]; // Optimize shared memory usage

    if (ty < 16 && tx == 0 && yIndex < height) { // Reduce if-statement complexity
        shared_vec[ty] = norm[yIndex];
    }
    __syncthreads();

    if (tx < width && xIndex < width && yIndex < height) { // Simplify bounds check
        array[yIndex * pitch + xIndex] += shared_vec[ty];
    }
}