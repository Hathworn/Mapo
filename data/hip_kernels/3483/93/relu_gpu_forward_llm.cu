#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Use shared memory to reduce global memory accesses
    extern __shared__ float shared_in[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;
    
    // Load data into shared memory
    if (tid < N) {
        shared_in[local_tid] = in[tid];
        __syncthreads(); // Synchronize threads within a block
        out[tid] = shared_in[local_tid] > 0 ? shared_in[local_tid] : 0;
    }
}