#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory to improve memory access efficiency
    __shared__ float indata_shared[BLOCK_SIZE];
    __shared__ float outgrad_shared[BLOCK_SIZE];
    
    if (tid < N) {
        indata_shared[threadIdx.x] = indata[tid];
        outgrad_shared[threadIdx.x] = outgrad[tid];
        __syncthreads(); // Ensure all threads have loaded their data
        
        // Perform computation using shared memory
        ingrad[tid] = indata_shared[threadIdx.x] > 0 ? outgrad_shared[threadIdx.x] : 0;
    }
}