#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void matrixEuclideanDistanceKernelFastPartialOut(float* in, float* out, int_least64_t n, int_least64_t m, int_least64_t start_out, int_least64_t end_out) {
    __shared__ float Ys[16][16];
    __shared__ float Xs[16][16];
    
    int_least64_t bx = blockIdx.x, by = blockIdx.y;
    int_least64_t tx = threadIdx.x, ty = threadIdx.y;

    int_least64_t yBegin = by * 16 * m;
    int_least64_t xBegin = bx * 16 * m;
    
    int_least64_t yEnd = yBegin + m - 1;
    float s = 0.0f;

    // Loop updated to fetch elements efficiently
    for (int_least64_t y = yBegin, x = xBegin; y <= yEnd; y += 16, x += 16) {
        Ys[ty][tx] = in[y + ty * m + tx];
        Xs[tx][ty] = in[x + ty * m + tx];
        __syncthreads();
        
        // Minimize index calculations inside the loop
        #pragma unroll
        for (int_least64_t k = 0; k < 16; ++k) {
            float diff = Ys[ty][k] - Xs[k][tx];
            s += diff * diff;
        }
        __syncthreads();
    }

    // Calculate output index and conditional check
    int_least64_t o = by * 16 * n + ty * n + bx * 16 + tx;
    if (o >= start_out && o < end_out) {
        out[o - start_out] = s;
    }
}