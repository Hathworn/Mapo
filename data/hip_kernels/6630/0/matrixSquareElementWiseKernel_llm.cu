#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__device__ int getGlobalIdx_2D_2D()
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y)
                 + (threadIdx.y * blockDim.x)
                 + threadIdx.x;
    return threadId;
}

__global__ void matrixSquareElementWiseKernel(float* in, float* out, int n, int m){
    // Calculate unique index for this thread
    int index = blockIdx.y * gridDim.x * blockDim.x * blockDim.y + blockIdx.x * blockDim.x * blockDim.y +
                threadIdx.y * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds before operation
    if (index < n*m){
        // Perform square operation
        out[index] = in[index] * in[index];
    }
}