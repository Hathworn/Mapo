#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void matrixEuclideanDistanceKernelFastPartialOut(float* in_X, float* in_Y, float* out, int_least64_t n, int_least64_t m, int_least64_t start_out, int_least64_t end_out) {
    __shared__ float Ys[16][16];
    __shared__ float Xs[16][16];

    int_least64_t bx = blockIdx.x, by = blockIdx.y;
    int_least64_t tx = threadIdx.x, ty = threadIdx.y;

    int_least64_t yBegin = by * 16 * m;
    int_least64_t xBegin = bx * 16 * m;

    int_least64_t yEnd = yBegin + m - 1;
    int_least64_t o = by * 16 * n + ty * n + bx * 16 + tx;

    float s = 0;

    // Loop unrolling for better performance
    for (int_least64_t k = 0; k < m; k += 16) {
        if (k + tx < m && by * 16 + ty < 16) {
            Ys[ty][tx] = in_Y[yBegin + k + ty * m + tx];
            Xs[ty][tx] = in_X[xBegin + k + ty * m + tx];
        }
        __syncthreads();

        #pragma unroll
        for (int_least64_t i = 0; i < 16; i++) {
            float tmp = Ys[ty][i] - Xs[i][tx];
            s += tmp * tmp;
        }
        __syncthreads();
    }

    if (o >= start_out && o < end_out) {
        out[o - start_out] = s;
    }
}