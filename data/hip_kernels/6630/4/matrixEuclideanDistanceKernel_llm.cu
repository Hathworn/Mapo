#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void matrixEuclideanDistanceKernel(float* in, float* out, int n, int m) {
    extern __shared__ float Rs[];
    float tmp, s;
    int myRow = blockIdx.x * blockDim.x + threadIdx.x;

    for (int r = 0; r < n; r++) { // outer loop
        s = 0;
        int myIdx = threadIdx.x;

        // Unroll the shared memory load loop
        for (int i = myIdx; i < m; i += blockDim.x) {
            Rs[i] = in[r * m + i];
        }

        __syncthreads();

        // Optimize by fully unrolling the loop, calculation only for myRow < n
        if (myRow < n) {
            for (int i = 0; i < m; i++) {
                tmp = Rs[i] - in[myRow * m + i];
                s += tmp * tmp;
            }
            out[myRow * n + r] = s;
        }

        __syncthreads();
    }
}