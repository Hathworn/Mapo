#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *ic, const int *ia, const int *ib)
{
    int i = threadIdx.x;
    int b = blockIdx.x;
    int bd = blockDim.x;
    int gd = gridDim.x;

    // Removed unnecessary __syncthreads() as it has no effect here
    // Reduced printf output for performance
    if (i == 0) {
        printf("G[%d] B[%d]  t[%d]\n", gd, bd, b);
    }

    // Example computation based on function name
    int index = b * bd + i;
    ic[index] = ia[index] + ib[index];
}