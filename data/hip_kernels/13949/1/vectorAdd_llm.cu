#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    // Use shared memory to optimize data access
    extern __shared__ float shared_mem[];
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        // Load inputs to shared memory
        shared_mem[threadIdx.x] = A[i] + B[i];
        __syncthreads();
        // Write result from shared memory
        C[i] = shared_mem[threadIdx.x];
    }
}