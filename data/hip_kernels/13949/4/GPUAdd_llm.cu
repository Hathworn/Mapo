#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GPUAdd(float *array1, float *array2, float *result, int WIDTH)
{
    // Calculate global index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Perform out-of-bounds check
    if (i < WIDTH) {
        result[i] = array1[i] + array2[i];
    }
}