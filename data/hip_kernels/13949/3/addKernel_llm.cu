#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 256

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Use blockIdx.x and blockDim.x for multi-block parallel processing
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if index is within bounds
    if (i < BLOCK_SIZE) {
        c[i] = a[i] + b[i];
    }
}