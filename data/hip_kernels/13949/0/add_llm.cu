#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c)
{
    // Use threadIdx.x to write to correct index in c
    int idx = threadIdx.x;
    if (idx == 0) {
        // Only thread 0 computes the result
        *c = a + b;
    }
}