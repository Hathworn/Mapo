#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GPUMult(int *A, int *B, int *C, int WIDTH)
{
    int sol = 0;
    int i = threadIdx.x + blockIdx.x * blockDim.x;  // Calculate global row index
    int j = threadIdx.y + blockIdx.y * blockDim.y;  // Calculate global column index

    if (i < WIDTH && j < WIDTH) {
        for (int k = 0; k < WIDTH; k++) {
            sol += A[j * WIDTH + k] * B[k * WIDTH + i];
        }
        C[j * WIDTH + i] = sol;
    }
}