#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloGPU()
{
    // Use warp synchronous printf for optimization
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid == 0) // Limit printing to the first thread
    {
        printf("Hello from the GPU.\n");
    }
}