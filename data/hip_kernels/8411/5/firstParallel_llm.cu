#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void firstParallel()
{
    // Use blockIdx, blockDim, and threadIdx for better parallel execution
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure only one thread performs the print operation to avoid clutter
    if (idx == 0) {
        printf("This should be running in parallel.\n");
    }
}