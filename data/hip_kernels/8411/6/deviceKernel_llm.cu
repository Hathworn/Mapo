#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void deviceKernel(int *a, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory to cache data between threads if necessary
    for (int i = idx; i < N; i += stride)
    {
        // Check boundary conditions to prevent out-of-bounds access
        if (i < N)
        {
            a[i] = 1;
        }
    }
}