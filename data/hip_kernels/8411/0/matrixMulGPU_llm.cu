#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N  64

__global__ void matrixMulGPU(int *a, int *b, int *c)
{
    // Utilize shared memory for performance optimization
    __shared__ int shared_a[N][N];
    __shared__ int shared_b[N][N];

    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;
    int val = 0;

    for (int k = 0; k < N; k += blockDim.x) {
        // Load data into shared memory
        if (row < N && (k + threadIdx.y) < N)
            shared_a[threadIdx.x][threadIdx.y] = a[row * N + k + threadIdx.y];
        
        if (col < N && (k + threadIdx.x) < N)
            shared_b[threadIdx.x][threadIdx.y] = b[(k + threadIdx.x) * N + col];
        
        __syncthreads();

        // Accumulate product results
        for (int n = 0; n < blockDim.x; ++n)
            val += shared_a[threadIdx.x][n] * shared_b[n][threadIdx.y];
        
        __syncthreads();
    }

    if (row < N && col < N)
        c[row * N + col] = val;
}