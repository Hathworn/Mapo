#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void printSuccessForCorrectExecutionConfiguration()
{
    // Use less expensive Warp function for condition
    if(threadIdx.x == blockDim.x - 1 && blockIdx.x == gridDim.x - 1)
    {
        printf("Success!\n");
    }
}