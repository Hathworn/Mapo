#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addVectorsInto(float *result, float *a, float *b, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < N) {
        result[idx] = a[idx] + b[idx]; // Perform addition for valid index
    }
}