#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void loop()
{
    // Use local variable to store thread ID to avoid repeated access to threadIdx.x
    int thread_id = threadIdx.x;
    printf("This is iteration number %d\n", thread_id);
}