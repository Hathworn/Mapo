#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initializeElementsTo(int initialValue, int *a, int N)
{
    // Use block size that is a multiple of warp size for better efficiency
    const int stride = blockDim.x * gridDim.x;
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < N; i += stride) {
        a[i] = initialValue;
    }
}