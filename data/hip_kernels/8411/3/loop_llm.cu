#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void loop()
{
    // Calculate global thread index
    int globalIndex = threadIdx.x + blockIdx.x * blockDim.x;

    // Print the iteration number using the calculated global index
    printf("This is iteration number %d\n", globalIndex);
}