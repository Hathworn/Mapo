#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloGPU()
{
    // Use thread 0 in block 0 to minimize redundant executions
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Hello from the GPU.\n");
    }
}