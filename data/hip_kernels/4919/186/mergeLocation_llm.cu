#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Use shared memory for improved memory access efficiency
    __shared__ short2 sharedLoc[256]; // Adjust size based on blockDim.x

    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Load data into shared memory
        sharedLoc[threadIdx.x] = loc_[ptidx];
        __syncthreads();

        // Use cached shared data
        x[ptidx] = sharedLoc[threadIdx.x].x * scale;
        y[ptidx] = sharedLoc[threadIdx.x].y * scale;
    }
}