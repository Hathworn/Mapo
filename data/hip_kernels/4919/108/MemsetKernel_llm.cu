#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Use cooperative grid-stride loop for improved memory access pattern
    for (int row = i; row < h; row += gridDim.y * blockDim.y) {
        for (int col = j; col < w; col += gridDim.x * blockDim.x) {
            int pos = row * w + col;
            image[pos] = value;
        }
    }
}