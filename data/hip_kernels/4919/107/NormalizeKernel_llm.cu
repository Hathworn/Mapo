#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Calculate row index
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Calculate column index

    if (i < h && j < w)  // Combined boundary check
    {
        const int pos = i * s + j; // Compute the linear index

        float scale = normalization_factor[pos]; // Access normalization factor

        float invScale = (scale == 0.0f) ? 1.0f : __frcp_rd(scale); // Use fast reciprocal

        image[pos] *= invScale; // Apply normalization
    }
}