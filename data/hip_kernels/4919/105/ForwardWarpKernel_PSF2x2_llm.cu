#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Pre-compute time scaled positions
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    float px, py;
    float dx = modff (cx, &px);
    float dy = modff (cy, &py);

    // Cast to integer coordinates
    int tx = (int) px;
    int ty = (int) py;
    float value = src[image_row_offset + j];

    // Precompute weights for efficiency
    float w_br = dx * dy; // bottom right
    float w_bl = (1.0f - dx) * dy; // bottom left
    float w_ul = (1.0f - dx) * (1.0f - dy); // upper left
    float w_ur = dx * (1.0f - dy); // upper right
    
    // Accumulate results using precomputed weights
    if (tx < w && tx >= 0 && ty < h && ty >= 0) {
        _atomicAdd(dst + ty * image_stride + tx, value * w_br);
        _atomicAdd(normalization_factor + ty * image_stride + tx, w_br);
    }

    tx -= 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0) {
        _atomicAdd(dst + ty * image_stride + tx, value * w_bl);
        _atomicAdd(normalization_factor + ty * image_stride + tx, w_bl);
    }

    ty -= 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0) {
        _atomicAdd(dst + ty * image_stride + tx, value * w_ul);
        _atomicAdd(normalization_factor + ty * image_stride + tx, w_ul);
    }

    tx += 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0) {
        _atomicAdd(dst + ty * image_stride + tx, value * w_ur);
        _atomicAdd(normalization_factor + ty * image_stride + tx, w_ur);
    }
}