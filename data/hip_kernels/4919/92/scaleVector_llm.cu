#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Using block size optimization to avoid divergence
    const int stride = blockDim.x * gridDim.x;
    for (int pos = blockIdx.x * blockDim.x + threadIdx.x; pos < len; pos += stride)
    {
        d_res[pos] = d_src[pos] * scale;
    }
}