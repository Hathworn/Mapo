#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds and early return optimization
    if (i >= h || j >= w) return;

    const int pos = i * s + j;

    // Use conditional operator to reduce divergence
    float scale = normalization_factor[pos];
    image[pos] *= (scale == 0.0f) ? 1.0f : (1.0f / scale);
}