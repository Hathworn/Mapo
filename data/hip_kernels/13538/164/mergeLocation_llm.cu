#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate global thread index
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce global memory access
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];
        float scaledX = loc.x * scale;
        float scaledY = loc.y * scale;

        __syncthreads(); // Ensure all threads have completed computation before writing back

        x[ptidx] = scaledX;
        y[ptidx] = scaledY;
    }
}