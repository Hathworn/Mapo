#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matAdd(int *a, int *b, int *c)
{
    // Calculate global thread index directly using grid and block dimensions
    int tID = blockIdx.z * blockDim.x * blockDim.y * gridDim.x * gridDim.y
            + blockIdx.y * blockDim.x * blockDim.y * gridDim.x
            + blockIdx.x * blockDim.x * blockDim.y
            + threadIdx.z * blockDim.x * blockDim.y
            + threadIdx.y * blockDim.x
            + threadIdx.x;

    c[tID] = a[tID] + b[tID];
}