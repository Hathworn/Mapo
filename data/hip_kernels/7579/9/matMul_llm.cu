#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matMul(double *a, double *b, double *c)
{
    // Use shared memory to reduce global memory accesses
    __shared__ double sharedA[BLOCK_SIZE]; 
    __shared__ double sharedB[BLOCK_SIZE]; 

    const int NUM_THREAD_IN_BLOCK = blockDim.x * blockDim.y * blockDim.z;

    int bID = blockIdx.z * (gridDim.y * gridDim.x * NUM_THREAD_IN_BLOCK) + blockIdx.y * (gridDim.x * NUM_THREAD_IN_BLOCK) + (blockIdx.x * (blockDim.x * blockDim.y * blockDim.z));
    int tID = bID + ((blockDim.y * blockDim.x) * threadIdx.z) + (blockDim.x * threadIdx.y) + threadIdx.x;

    double sum = 0.0;

    // Loop over tiles
    for (int tile = 0; tile < MATRIX_J / BLOCK_SIZE; ++tile) {
        // Load elements into shared memory
        sharedA[threadIdx.x] = a[tID * MATRIX_J + tile * BLOCK_SIZE + threadIdx.x];
        sharedB[threadIdx.x] = b[(tile * BLOCK_SIZE + threadIdx.x) * MATRIX_J + tID];

        __syncthreads(); // Ensure all loads are complete

        // Compute partial result
        for (int i = 0; i < BLOCK_SIZE; i++) {
            sum += sharedA[i] * sharedB[i];
        }

        __syncthreads(); // Ensure all computations are complete
    }

    c[tID] = sum;
}