#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matMul(float *a, float *b, float *c, int M, int N, int K)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if(row >= M || col >= N)
        return;

    float sum = 0.f;

    // Optimize loop for better memory access patterns
    for(int k = 0; k < K; k++)
    {
        sum += a[row * K + k] * b[k * N + col];
    }

    c[row * N + col] = sum;
}