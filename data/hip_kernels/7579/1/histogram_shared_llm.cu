#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DATA_SIZE (1024 * 1024 * 256)
#define DATA_RANGE (256)

void printHist(int * arr, char * str);

__global__ void histogram_shared(float * a, int * histo, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int sh[DATA_RANGE];

    // Initialize shared memory with one warp
    for (int i = threadIdx.x; i < DATA_RANGE; i += blockDim.x) {
        sh[i] = 0;
    }
    __syncthreads();

    // Accumulate histogram counts in shared memory
    if (tid < n) {
        atomicAdd(&sh[(int)a[tid]], 1);
    }
    __syncthreads();

    // Write shared memory counts to global memory
    for (int i = threadIdx.x; i < DATA_RANGE; i += blockDim.x) {
        atomicAdd(&histo[i], sh[i]);
    }
}