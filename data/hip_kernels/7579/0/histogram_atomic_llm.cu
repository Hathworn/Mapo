#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DATA_SIZE (1024 * 1024 * 256)
#define DATA_RANGE (256)

void printHist(int * arr, char * str);

__global__ void histogram_atomic(float * a, int * histo, int n) {
    // Calculate the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if (tid < n) {
        // Use atomic operation to safely update histogram
        atomicAdd(&histo[(int)a[tid]], 1);
    }
}