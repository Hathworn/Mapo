#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CudaKernel_BatchResize_GRAY2GRAY( int src_width, unsigned char* src_image, int num_rects, int* rects, int dst_width, int dst_height, float* dst_ptr )
{
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    const int dst_image_size = dst_width * dst_height;
    if (num_rects * dst_image_size <= gid) {
        return;
    }

    const int image_index = gid / dst_image_size;
    const int pixel_index = gid % dst_image_size;

    // Calculate scale and coordinate transformation variables
    const float scale_x = (float)(rects[image_index * 4 + 2]) / dst_width;
    const float scale_y = (float)(rects[image_index * 4 + 3]) / dst_height;
    const float fx = (float)((pixel_index % dst_width) + 0.5f) * scale_x - 0.5f;
    const float fy = (float)((pixel_index / dst_width) + 0.5f) * scale_y - 0.5f;
    
    const int coor_x_in_rect = floor(fx);
    const int coor_y_in_rect = floor(fy);
    
    const float fract_x = fx - coor_x_in_rect;
    const float fract_y = fy - coor_y_in_rect;

    // Precompute fixed values
    const int src_x = rects[image_index * 4 + 0];
    const int src_y = rects[image_index * 4 + 1];
    const int base_index = src_width * (src_y + coor_y_in_rect) + src_x + coor_x_in_rect;

    // Use bilinear interpolation with optimized indexing
    float value = 0.0f;
    value += src_image[base_index] * (1.0f - fract_x) * (1.0f - fract_y);
    value += src_image[base_index + 1] * fract_x * (1.0f - fract_y);
    value += src_image[base_index + src_width] * (1.0f - fract_x) * fract_y;
    value += src_image[base_index + src_width + 1] * fract_x * fract_y;

    dst_ptr[gid] = value / 255.0f;  // Normalize and store result
}