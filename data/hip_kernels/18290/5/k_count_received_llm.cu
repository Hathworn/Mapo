#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void k_count_received(int nr_total_blocks, uint* d_n_recv_by_block, uint* d_spine_cnts)
{
    int bid = blockIdx.x * blockDim.x + threadIdx.x;

    if (bid < nr_total_blocks) {
        // Coalesce memory access by using stride of 1
        uint recv_count = d_n_recv_by_block[bid];
        d_spine_cnts[bid * 10 + CUDA_BND_S_NEW] = recv_count;
    }
}