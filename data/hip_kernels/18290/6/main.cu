#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "k_reorder_send_buf_total.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    int nr_prts = 1;
int nr_total_blocks = 1;
uint *d_bidx = NULL;
hipMalloc(&d_bidx, XSIZE*YSIZE);
uint *d_sums = NULL;
hipMalloc(&d_sums, XSIZE*YSIZE);
float4 *d_xi4 = NULL;
hipMalloc(&d_xi4, XSIZE*YSIZE);
float4 *d_pxi4 = NULL;
hipMalloc(&d_pxi4, XSIZE*YSIZE);
float4 *d_xchg_xi4 = NULL;
hipMalloc(&d_xchg_xi4, XSIZE*YSIZE);
float4 *d_xchg_pxi4 = NULL;
hipMalloc(&d_xchg_pxi4, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
k_reorder_send_buf_total<<<gridBlock, threadBlock>>>(nr_prts,nr_total_blocks,d_bidx,d_sums,d_xi4,d_pxi4,d_xchg_xi4,d_xchg_pxi4);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
k_reorder_send_buf_total<<<gridBlock, threadBlock>>>(nr_prts,nr_total_blocks,d_bidx,d_sums,d_xi4,d_pxi4,d_xchg_xi4,d_xchg_pxi4);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
k_reorder_send_buf_total<<<gridBlock, threadBlock>>>(nr_prts,nr_total_blocks,d_bidx,d_sums,d_xi4,d_pxi4,d_xchg_xi4,d_xchg_pxi4);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}