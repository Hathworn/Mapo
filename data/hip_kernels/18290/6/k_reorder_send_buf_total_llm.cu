#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void k_reorder_send_buf_total(int nr_prts, int nr_total_blocks, uint* d_bidx, uint* d_sums, float4* d_xi4, float4* d_pxi4, float4* d_xchg_xi4, float4* d_xchg_pxi4)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Return early if thread index exceeds total particles
    if (i >= nr_prts)
        return;

    // Avoid unnecessary memory accesses by checking the condition first
    if (d_bidx[i] == CUDA_BND_S_OOB) {
        int j = d_sums[i];
        
        // Use one memory transaction per float4 element
        d_xchg_xi4[j] = d_xi4[i];
        d_xchg_pxi4[j] = d_pxi4[i];
    }
}