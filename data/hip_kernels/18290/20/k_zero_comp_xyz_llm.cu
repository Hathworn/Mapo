#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void k_zero_comp_xyz(float *data, uint n, uint stride)
{
    // Optimize memory access by using a grid-stride loop
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint p = blockIdx.y * stride;
    
    for (uint i = idx; i < n; i += blockDim.x * gridDim.x) {
        data[i + p] = 0.f;
    }
}