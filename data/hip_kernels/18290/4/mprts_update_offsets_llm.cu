#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void mprts_update_offsets(int nr_total_blocks, uint* d_off, uint* d_spine_sums)
{
    int bid = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;

    // Optimization: Use early return to avoid extra branching
    if (bid > nr_total_blocks) return;

    // Fetch data from global memory efficiently
    int offset_index = bid * CUDA_BND_STRIDE;
    d_off[bid] = d_spine_sums[offset_index];
}