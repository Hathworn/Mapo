#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_local_time(int *next, double *local_time, double GTIME) {
    unsigned int gtid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use __syncthreads() to ensure all threads have updated 'who'
    __shared__ int shared_next[256];
    __shared__ double shared_local_time[256];

    // Load data into shared memory for coalesced reads
    shared_next[threadIdx.x] = next[gtid];
    __syncthreads();

    int who = shared_next[threadIdx.x];

    if (who < 0) return;

    // Avoid unnecessary global memory write
    shared_local_time[threadIdx.x] = GTIME;
    local_time[who] = shared_local_time[threadIdx.x];
}