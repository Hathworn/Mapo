#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reposition(double4 *ac, double4 *ac1, double4 *ac2, double4 *af, unsigned long nextsize)
{
    // Calculate the global index once to avoid recomputation
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single if statement to reduce branching
    if (i < nextsize) {
        // Perform memory accesses in a coalesced manner to improve performance
        double4 temp_ac = ac[i];
        double4 temp_ac1 = ac1[i];
        double4 temp_ac2 = ac2[i];

        af[i] = temp_ac;
        af[i + nextsize] = temp_ac1;
        af[i + 2 * nextsize] = temp_ac2;
    }
}