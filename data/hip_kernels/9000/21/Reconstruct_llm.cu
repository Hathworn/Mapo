#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Reconstruct(int *nex, unsigned long nextsize, double4 *pc, double4 *vc, double4 *a3, double4 *a, double4 *a1, double4 *a2, double4 *pva3, double4 *aaa) {

    unsigned int gtid = blockIdx.x * blockDim.x + threadIdx.x;

    // Precompute index to reduce repeated computation
    if (gtid < 6 * nextsize) {
        int k = gtid / nextsize;
        int who = nex[gtid % nextsize];  // Use modular arithmetic for cleaner index calculation
        if (k == 0) {
            pc[who] = pva3[gtid];
        } else if (k == 1) {
            vc[who] = pva3[gtid];
        } else if (k == 2) {
            a3[who] = pva3[gtid];
        } else if (k == 3) {
            a[who] = aaa[gtid - 3 * nextsize];
        } else if (k == 4) {
            a1[who] = aaa[gtid - 3 * nextsize];
        } else if (k == 5) {
            a2[who] = aaa[gtid - 3 * nextsize];
        }
    }
}