#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce(double4 *ac, double4 *ac1, double4 *ac2, unsigned int bf_real, unsigned int dimension) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = dimension * bf_real;
    double4 myacc;

    extern __shared__ double4 shaccelerations[];
    double4 *shacc = (double4*)shaccelerations;

    if (i < 3 * k) {
        // Reduce the number of branches by grouping them
        if (i < k) {
            myacc = ac[i];
        } else if (i < 2 * k) {
            myacc = ac1[i - k];
        } else {
            myacc = ac2[i - 2 * k];
        }

        // Load into shared memory and perform the reduction
        shacc[threadIdx.x] = (i < k) ? ac[i + k] : ((i < 2 * k) ? ac1[i] : ac2[i - k]);

        // Accumulate results
        myacc.x += shacc[threadIdx.x].x;
        myacc.y += shacc[threadIdx.x].y;
        myacc.z += shacc[threadIdx.x].z;

        // Write back results based on range
        if (i < k) {
            ac[i] = myacc;
        } else if (i < 2 * k) {
            ac1[i - k] = myacc;
        } else {
            ac2[i - 2 * k] = myacc;
        }
    }
}