#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dev_get_gravity_at_point(float eps2, float *eps, float *xh, float *yh, float *zh, float *xt, float *yt, float *zt, float *ax, float *ay, float *az, int n, float *field_m, float *fxh, float *fyh, float *fzh, float *fxt, float *fyt, float *fzt, int n_field) {
    // Calculate thread id
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Loop through particles, stride by blockDim * gridDim for efficiency
    while (tid < n) {
        float eps2_total = eps2 + eps[tid] * eps[tid];
        float ax_temp = 0.0f;
        float ay_temp = 0.0f;
        float az_temp = 0.0f;
        
        // Main loop for gravitational field calculation
        for (int i = 0; i < n_field; i++) {
            float dx = (fxh[i] - xh[tid]) + (fxt[i] - xt[tid]);
            float dy = (fyh[i] - yh[tid]) + (fyt[i] - yt[tid]);
            float dz = (fzh[i] - zh[tid]) + (fzt[i] - zt[tid]);
            float dr2 = dx * dx + dy * dy + dz * dz;
            
            if (dr2 > 0.0f) {
                float r2 = eps2_total + dr2;
                float inv_r = rsqrtf(r2);
                float tmp = field_m[i] * inv_r * inv_r * inv_r; // Use rsqrtf for better performance
                ax_temp += tmp * dx;
                ay_temp += tmp * dy;
                az_temp += tmp * dz;
            }
        }
        
        // Accumulate into global memory
        ax[tid] = ax_temp;
        ay[tid] = ay_temp;
        az[tid] = az_temp;
        
        // Proceed to the next thread within grid
        tid += blockDim.x * gridDim.x;
    }
}