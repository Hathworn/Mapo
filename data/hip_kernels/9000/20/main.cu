#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "Corrector_gpu.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    double GTIME = 1;
double *local_time = NULL;
hipMalloc(&local_time, XSIZE*YSIZE);
double *step = NULL;
hipMalloc(&step, XSIZE*YSIZE);
int *next = NULL;
hipMalloc(&next, XSIZE*YSIZE);
unsigned long nextsize = 1;
double4 *pos_CH = NULL;
hipMalloc(&pos_CH, XSIZE*YSIZE);
double4 *vel_CH = NULL;
hipMalloc(&vel_CH, XSIZE*YSIZE);
double4 *a_tot_D = NULL;
hipMalloc(&a_tot_D, XSIZE*YSIZE);
double4 *a1_tot_D = NULL;
hipMalloc(&a1_tot_D, XSIZE*YSIZE);
double4 *a2_tot_D = NULL;
hipMalloc(&a2_tot_D, XSIZE*YSIZE);
double4 *a_H0 = NULL;
hipMalloc(&a_H0, XSIZE*YSIZE);
double4 *a3_H = NULL;
hipMalloc(&a3_H, XSIZE*YSIZE);
double ETA6 = 1;
double ETA4 = 1;
double DTMAX = 1;
double DTMIN = 1;
unsigned int N = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
Corrector_gpu<<<gridBlock, threadBlock>>>(GTIME,local_time,step,next,nextsize,pos_CH,vel_CH,a_tot_D,a1_tot_D,a2_tot_D,a_H0,a3_H,ETA6,ETA4,DTMAX,DTMIN,N);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
Corrector_gpu<<<gridBlock, threadBlock>>>(GTIME,local_time,step,next,nextsize,pos_CH,vel_CH,a_tot_D,a1_tot_D,a2_tot_D,a_H0,a3_H,ETA6,ETA4,DTMAX,DTMIN,N);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
Corrector_gpu<<<gridBlock, threadBlock>>>(GTIME,local_time,step,next,nextsize,pos_CH,vel_CH,a_tot_D,a1_tot_D,a2_tot_D,a_H0,a3_H,ETA6,ETA4,DTMAX,DTMIN,N);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}