#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Corrector_gpu(double GTIME, double *local_time, double *step, int *next, unsigned long nextsize, double4 *pos_CH, double4 *vel_CH, double4 *a_tot_D, double4 *a1_tot_D, double4 *a2_tot_D, double4 *a_H0, double4 *a3_H, double ETA6, double ETA4, double DTMAX, double DTMIN, unsigned int N) {

    unsigned int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (gtid >= nextsize) return; // Boundary check
    
    // Load indices based on next and gtid
    int who = next[gtid];
    int who1 = gtid + nextsize;
    int who2 = who1 + nextsize;

    // Pre-calculate based on the above
    double h = GTIME - local_time[who];
    local_time[who] = GTIME;
    double h1 = 0.5 * h;
    double h2 = h1 * h1;
    double h3 = 0.75 / (h1 * h1 * h1);
    double h4 = 1.5 / (h2 * h2);
    double h5 = 7.5 / (h2 * h2 * h1);
    double over = 1.0 / 15.0;

    // Precomputing squares for future calculations
    for (int i = 0; i < 3; ++i) {
        int offset = i * nextsize;
        a_H0[gtid + offset].w = a_H0[gtid + offset].x * a_H0[gtid + offset].x +
                                a_H0[gtid + offset].y * a_H0[gtid + offset].y +
                                a_H0[gtid + offset].z * a_H0[gtid + offset].z;
    }

    double *A_H0_w = &a_H0[gtid].w;
    
    // Function to calculate each dimension
    auto CalculateDimension = [&] (double &pos, double &vel, int index, double &a3, double &a4half, double &a5half) {

	// Pre-calculation of values
        double Amin = a_H0[gtid].x - a_tot_D[who].x;
        double Aplu = a_H0[gtid].x + a_tot_D[who].x;
        double Jmin = h1 * (a_H0[who1].x - a1_tot_D[who].x);
        double Jplu = h1 * (a_H0[who1].x + a1_tot_D[who].x);
        double Smin = h2 * (a_H0[who2].x - a2_tot_D[who].x);
        double Splu = h2 * (a_H0[who2].x + a2_tot_D[who].x);
        
        // Corrector calculations
        pos = pos + h1 * vel - 0.4 * h2 * Amin + over * h2 * Jplu;
        vel = vel + h1 * Aplu - 0.4 * h1 * Jmin + over * h1 * Splu;
        pos += h1 * vel;
        
        // Higher order calculations for corrector step
        a3 = h3 * (-5.0 * Amin + 5.0 * Jplu - Smin);
        a4half = h4 * (-Jmin + Splu);
        a5half = h5 * (3.0 * Amin - 3.0 * Jplu + Smin);
        
        // Summation of higher-order terms
        a3 += h1 * a4half + 0.5 * h2 * a5half;
        a4half += h1 * a5half;
    };

    double a4halfx, a4halfy, a4halfz;
    double a5halfx, a5halfy, a5halfz;

    // Calculate for each of x, y, z dimensions
    CalculateDimension(pos_CH[who].x, vel_CH[who].x, 0, a3_H[who].x, a4halfx, a5halfx);
    CalculateDimension(pos_CH[who].y, vel_CH[who].y, 1, a3_H[who].y, a4halfy, a5halfy);
    CalculateDimension(pos_CH[who].z, vel_CH[who].z, 2, a3_H[who].z, a4halfz, a5halfz);

    // Final calculations
    a3_H[who].w = sqrt(a3_H[who].x * a3_H[who].x + a3_H[who].y * a3_H[who].y + a3_H[who].z * a3_H[who].z);
    double a4mod = sqrt(a4halfx * a4halfx + a4halfy * a4halfy + a4halfz * a4halfz);
    double a5mod = sqrt(a5halfx * a5halfx + a5halfy * a5halfy + a5halfz * a5halfz);

    // Timesteps with pre-calculated values
    double dt6 = (sqrt(A_H0_w[0] * A_H0_w[who2]) + A_H0_w[who1]) / (a5mod * a3_H[who].w + a4mod * a4mod);
    dt6 = ETA6 * pow(dt6, 1.0 / 6.0);

    double stp = h;
    double overh3 = 1.0 / (stp * stp * stp);
    double overh2 = 1.0 / (stp * stp);

    // Calculate a2 and a3 derivatives
    auto CalculateAccDerivative = [&] (double compTot, double comp1Tot, double &a2, double &a3) {
        a2 = overh2 * (-6.0 * (compTot - a_H0[gtid].x) - stp * (4.0 * a_H0[who1].x + 2.0 * comp1Tot));
        a3 = overh3 * (12.0 * (compTot - a_H0[gtid].x) + 6.0 * stp * (a_H0[who1].x + comp1Tot));
        a2 += h * a3;
    };

    double a2dx, a2dy, a2dz;
    double a3dx, a3dy, a3dz;

    CalculateAccDerivative(a_tot_D[who].x, a1_tot_D[who].x, a2dx, a3dx);
    CalculateAccDerivative(a_tot_D[who].y, a1_tot_D[who].y, a2dy, a3dy);
    CalculateAccDerivative(a_tot_D[who].z, a1_tot_D[who].z, a2dz, a3dz);

    double a2mod_w = a2dx * a2dx + a2dy * a2dy + a2dz * a2dz;
    a_H0[who2].w = a2mod_w;
    a3_H[who].w = a3dx * a3dx + a3dy * a3dy + a3dz * a3dz;

    double dt4 = sqrt(ETA4 * (sqrt(A_H0_w[0] * a2mod_w) + A_H0_w[who1]) / (sqrt(A_H0_w[who1] * a3_H[who].w) + a2mod_w));

    // Calculate final time step value
    double dt = 0.5 * dt4 + 0.5 * dt6;

    double rest = GTIME / (2.0 * step[who]);
    rest = (double)((int)(rest)) - rest;
    
    // Step-size update logic
    if (dt > 2.0 * step[who] && rest == 0.0 && 2.0 * step[who] <= DTMAX)
        step[who] *= 2.0;
    else if (dt < 0.5 * step[who])
        step[who] *= 0.25;
    else if (dt < step[who])
        step[who] *= 0.5;

    // Ensuring minimum step size
    if (step[who] < DTMIN)
        step[who] = DTMIN;

    // Update acceleration vectors
    a_tot_D[who] = a_H0[gtid];
    a1_tot_D[who] = a_H0[who1];
    a2_tot_D[who] = a_H0[who2];
}