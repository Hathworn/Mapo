#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Predictor (const double TIME, double4 *p_pred, float4 *v_pred, float4 *a_pred, double4 *p_corr, double4 *v_corr, double *loc_time, double4 *acc, double4 *acc1, double4 *acc2, double4 *acc3, int istart, int* nvec, int ppgpus, unsigned int N) {
    // Calculate global index
    int i = blockIdx.x*blockDim.x + threadIdx.x + istart;
    int cost = ppgpus + istart;

    // Validate index and adjust
    if (i >= cost) {
        i = nvec[i - cost];
        if (i >= istart && i < cost)
            i = -1;
    }
    if (i < 0)
        return;

    // Precompute powers of timestep and their multipliers
    double timestep = TIME - loc_time[i];
    double t2 = 0.5 * timestep * timestep;
    double t3 = timestep * t2 * 0.3333333333333333;
    double t4 = t2 * t2 * 0.0833333333333333;
    double t5 = t4 * timestep * 0.2;

    // Load vectors into registers
    double4 mypcorr = p_corr[i];
    double4 myvcorr = v_corr[i];
    double4 myacc = acc[i];
    double4 myacc1 = acc1[i];
    double4 myacc2 = acc2[i];
    double4 myacc3 = acc3[i];

    // Compute predicted position
    double4 myppred;
    myppred.x = mypcorr.x + timestep * myvcorr.x + t2 * myacc.x + t3 * myacc1.x + t4 * myacc2.x + t5 * myacc3.x;
    myppred.y = mypcorr.y + timestep * myvcorr.y + t2 * myacc.y + t3 * myacc1.y + t4 * myacc2.y + t5 * myacc3.y;
    myppred.z = mypcorr.z + timestep * myvcorr.z + t2 * myacc.z + t3 * myacc1.z + t4 * myacc2.z + t5 * myacc3.z;

    // Save predicted position
    p_pred[i].x = myppred.x;
    p_pred[i].y = myppred.y;
    p_pred[i].z = myppred.z;

    // Compute predicted velocity
    float4 mypred;
    mypred.x = myvcorr.x + timestep * myacc.x + t2 * myacc1.x + t3 * myacc2.x + t4 * myacc3.x;
    mypred.y = myvcorr.y + timestep * myacc.y + t2 * myacc1.y + t3 * myacc2.y + t4 * myacc3.y;
    mypred.z = myvcorr.z + timestep * myacc.z + t2 * myacc1.z + t3 * myacc2.z + t4 * myacc3.z;

    // Save predicted velocity
    v_pred[i].x = mypred.x;
    v_pred[i].y = mypred.y;
    v_pred[i].z = mypred.z;

    // Compute predicted acceleration
    mypred.x = myacc.x + timestep * myacc1.x + t2 * myacc2.x + t3 * myacc3.x;
    mypred.y = myacc.y + timestep * myacc1.y + t2 * myacc2.y + t3 * myacc3.y;
    mypred.z = myacc.z + timestep * myacc1.z + t2 * myacc2.z + t3 * myacc3.z;

    // Save predicted acceleration
    a_pred[i].x = mypred.x;
    a_pred[i].y = mypred.y;
    a_pred[i].z = mypred.z;
}