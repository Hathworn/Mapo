#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "Predictor.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    const double TIME = 1;
double4 *p_pred = NULL;
hipMalloc(&p_pred, XSIZE*YSIZE);
float4 *v_pred = NULL;
hipMalloc(&v_pred, XSIZE*YSIZE);
float4 *a_pred = NULL;
hipMalloc(&a_pred, XSIZE*YSIZE);
double4 *p_corr = NULL;
hipMalloc(&p_corr, XSIZE*YSIZE);
double4 *v_corr = NULL;
hipMalloc(&v_corr, XSIZE*YSIZE);
double *loc_time = NULL;
hipMalloc(&loc_time, XSIZE*YSIZE);
double4 *acc = NULL;
hipMalloc(&acc, XSIZE*YSIZE);
double4 *acc1 = NULL;
hipMalloc(&acc1, XSIZE*YSIZE);
double4 *acc2 = NULL;
hipMalloc(&acc2, XSIZE*YSIZE);
double4 *acc3 = NULL;
hipMalloc(&acc3, XSIZE*YSIZE);
int istart = 1;
int *nvec = NULL;
hipMalloc(&nvec, XSIZE*YSIZE);
int ppgpus = 1;
unsigned int N = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
Predictor<<<gridBlock, threadBlock>>>(TIME,p_pred,v_pred,a_pred,p_corr,v_corr,loc_time,acc,acc1,acc2,acc3,istart,nvec,ppgpus,N);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
Predictor<<<gridBlock, threadBlock>>>(TIME,p_pred,v_pred,a_pred,p_corr,v_corr,loc_time,acc,acc1,acc2,acc3,istart,nvec,ppgpus,N);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
Predictor<<<gridBlock, threadBlock>>>(TIME,p_pred,v_pred,a_pred,p_corr,v_corr,loc_time,acc,acc1,acc2,acc3,istart,nvec,ppgpus,N);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}