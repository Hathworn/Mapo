#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_partial(double4 *a, double4 *b, unsigned int nextsize) {

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= nextsize) 
        return;

    extern __shared__ double4 shaccelerations[];
    double4 *shacc = (double4*) shaccelerations;

    // Load a[] into shared memory to allow coalesced global memory access
    shacc[threadIdx.x] = a[i];
    __syncthreads();  // Synchronize to ensure all threads have loaded their data

    double4 myacc = b[i];

    // Utilize shared memory - minimize repeated global memory accesses
    myacc.x += shacc[threadIdx.x].x;
    myacc.y += shacc[threadIdx.x].y;
    myacc.z += shacc[threadIdx.x].z;

    b[i] = myacc;
}