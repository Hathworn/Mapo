#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dev_get_potential_energy(float *partial_results, float eps2, float *field_m, float *fxh, float *fyh, float *fzh, float *fxt, float *fyt, float *fzt, int n_field) {
    extern __shared__ float thread_results[];
    unsigned int i, j;
    float dx, dy, dz, r, dr2, potential_energy = 0;

    // Use single loop with strided access to minimize divergence
    for (j = threadIdx.x + blockIdx.x * blockDim.x; j < n_field; j += blockDim.x * gridDim.x) {
        for (i = 0; i < n_field; i++) {
            if (i != j) { // Avoid self-interaction
                dx = (fxh[i] - fxh[j]) + (fxt[i] - fxt[j]);
                dy = (fyh[i] - fyh[j]) + (fyt[i] - fyt[j]);
                dz = (fzh[i] - fzh[j]) + (fzt[i] - fzt[j]);
                dr2 = dx * dx + dy * dy + dz * dz;
                r = rsqrtf(eps2 + dr2); // Use reciprocal square root for efficiency
                potential_energy -= field_m[i] * field_m[j] * r;
            }
        }
    }

    // Reduce results within block
    thread_results[threadIdx.x] = potential_energy;
    __syncthreads();
    for (i = blockDim.x / 2; i > 0; i >>= 1) {
        if (threadIdx.x < i) {
            thread_results[threadIdx.x] += thread_results[threadIdx.x + i];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        partial_results[blockIdx.x] = thread_results[0];
    }
}