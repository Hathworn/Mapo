#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_potential_gpu(float *m, float *x, float *y, float *z, float *phi, int N, int N1) {
    // Define shared memory for block calculations
    extern __shared__ float sharedMem[];
    float *sharedX = sharedMem;
    float *sharedY = sharedX + blockDim.x;
    float *sharedZ = sharedY + blockDim.x;

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    float xi, yi, zi, potential = 0.0f;
    
    if (i < (N1 == 0 ? N : N1)) {
        xi = x[i];
        yi = y[i];
        zi = z[i];
        
        for (int tile = 0; tile < (N + blockDim.x - 1) / blockDim.x; tile++) {
            int j = tile * blockDim.x + threadIdx.x;
            if (j < N) {
                // Load data into shared memory
                sharedX[threadIdx.x] = x[j];
                sharedY[threadIdx.x] = y[j];
                sharedZ[threadIdx.x] = z[j];
            }
            __syncthreads(); // Synchronize to ensure complete load
            
            for (int k = 0; k < blockDim.x; k++) {
                int idx = tile * blockDim.x + k;
                if (idx < N && i != idx) {
                    float rijx = xi - sharedX[k];
                    float rijy = yi - sharedY[k];
                    float rijz = zi - sharedZ[k];
                    potential -= m[idx] / sqrtf(rijx * rijx + rijy * rijy + rijz * rijz);
                }
            }
            __syncthreads(); // Synchronize to prepare for next tile
        }
        phi[i] = potential;
    }
}