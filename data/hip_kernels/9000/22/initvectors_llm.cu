#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initvectors(double4 *acc3, float4 *apred) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < gridDim.x * blockDim.x) { // Check to prevent out-of-bounds access
        acc3[i] = make_double4(0.0, 0.0, 0.0, acc3[i].w); // Initialize acc3 only x, y, z
        apred[i] = make_float4(0.0f, 0.0f, 0.0f, apred[i].w); // Initialize apred only x, y, z
    }
}