#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimized for parallel execution
__global__ void dev_get_potential_at_point( float eps2, float *eps, float *xh, float *yh, float *zh, float *xt, float *yt, float *zt, float *phi, int n, float *field_m, float *fxh, float *fyh, float *fzh, float *fxt, float *fyt, float *fzt, int n_field) {
    float dx, dy, dz, r, dr2, eps2_total;

    // Use shared memory for faster access
    extern __shared__ float shared_eps[];
    int local_tid = threadIdx.x;
    if (local_tid < n) {
        shared_eps[local_tid] = eps[local_tid];
    }
    __syncthreads();

    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < n; tid += blockDim.x * gridDim.x) {
        eps2_total = eps2 + shared_eps[threadIdx.x] * shared_eps[threadIdx.x];
        phi[tid] = 0;

        for (int i = 0; i < n_field; i++) {
            dx = (fxh[i] - xh[tid]) + (fxt[i] - xt[tid]);
            dy = (fyh[i] - yh[tid]) + (fyt[i] - yt[tid]);
            dz = (fzh[i] - zh[tid]) + (fzt[i] - zt[tid]);
            dr2 = dx * dx + dy * dy + dz * dz;

            if (dr2 > 0) {
                r = rsqrt(eps2_total + dr2);  // Use fast math function for inverse sqrt
                phi[tid] -= field_m[i] * r;  // Multiplication instead of division for efficiency
            }
        }
    }
}