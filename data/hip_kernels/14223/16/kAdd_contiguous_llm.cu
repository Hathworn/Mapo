#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Use shared memory for block-wise processing to improve memory access patterns
    __shared__ float s_a[256];
    __shared__ float s_b[256];

    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        s_a[threadIdx.x] = a[i];
        s_b[threadIdx.x] = b[i];
        __syncthreads(); // Ensure all threads have loaded before continuing

        dest[i] = s_a[threadIdx.x] + s_b[threadIdx.x];

        __syncthreads(); // Ensure all threads have written before next iteration
    }
}