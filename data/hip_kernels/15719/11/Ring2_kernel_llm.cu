#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void Ring2_kernel(float *A, float *BP, int *corrAB, int *mask, int *m, int ring, int c, int h, int w)
{
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;
    if (id1 < size && mask[id1] != 0) {  // Combine conditions for early exit
        int y2 = corrAB[2 * id1 + 1], x2 = corrAB[2 * id1 + 0];
        
        for (int dy = -ring; dy <= ring; dy++) {
            int _y2 = y2 + dy;
            if (_y2 >= 0 && _y2 < h) {  // Pre-check y-boundary condition
                for (int dx = -ring; dx <= ring; dx++) {
                    int _x2 = x2 + dx;
                    if (_x2 >= 0 && _x2 < w) {  // Check x-boundary condition
                        m[_y2 * w + _x2] = 1;  // Update m only when both conditions are satisfied
                    }
                }
            }
        }
    }

    return;
}