#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void blend_kernel(float *A, float *BP, float *M, float *AP, float alpha, int c, int h, int w)
{
    int _id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;
    if (_id < c * size)
    {
        int id = _id % size;
        int dc = _id / size;
        
        // Pre-calculate index to avoid repetition
        int index = dc * size + id;
        
        // Use ternary operator for more concise weight calculation
        float weight = (M[id] < 0.05f) ? 0.f : alpha;
        
        // Reduce repeated arithmetic operations
        AP[index] = A[index] * weight + BP[index] * (1.f - weight);
    }
    return;
}