#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void patchmatch2_argmax_kernel(float *conv, int *prev_corrAB_upsampled, int *corrAB, int s_rad, int c, int h, int w)
{
    int h1 = h, w1 = w;
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int size1 = h1 * w1;
    int s_size = 2 * s_rad + 1;

    if (id1 < size1) {
        float conv_max = -1;
        int x2 = prev_corrAB_upsampled[2 * id1];
        int y2 = prev_corrAB_upsampled[2 * id1 + 1];
        int best_x2 = x2, best_y2 = y2; // Store best result coordinates

        for (int dx2 = -s_rad; dx2 <= s_rad; dx2++) {
            for (int dy2 = -s_rad; dy2 <= s_rad; dy2++) {
                int new_x2 = x2 + dx2;
                int new_y2 = y2 + dy2;

                // Boundary check
                if (new_x2 >= 0 && new_x2 < w1 && new_y2 >= 0 && new_y2 < h1) {
                    int s_idx = (dy2 + s_rad) * s_size + (dx2 + s_rad);
                    float conv_result = conv[id1 * (s_size * s_size) + s_idx];
                    if (conv_result > conv_max) {
                        conv_max = conv_result;
                        best_x2 = new_x2;
                        best_y2 = new_y2;
                    }
                }
            }
        }
        
        // Use local variables to reduce global memory writes
        corrAB[id1 * 2] = best_x2;
        corrAB[id1 * 2 + 1] = best_y2;
    }
}