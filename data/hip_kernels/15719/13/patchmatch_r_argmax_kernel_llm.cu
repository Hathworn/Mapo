#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void patchmatch_r_argmax_kernel(
    float *conv, float *target, float *match, int *correspondence, 
    int c1, int h1, int w1, int h2, int w2 )
{
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int size1 = h1 * w1, size2 = h2 * w2;

    if (id1 < size1) {
        double conv_max = -1e20;
        int best_x2 = 0, best_y2 = 0; // Store best matches
        
        for (int id2 = 0; id2 < size2; ++id2) {
            float conv_result = conv[id1 * size2 + id2];
            
            if (conv_result > conv_max) {
                conv_max = conv_result;
                best_x2 = id2 % w2;  // Calculate x2
                best_y2 = id2 / w2;  // Calculate y2
            }
        }
        
        correspondence[id1 * 2 + 0] = best_x2;
        correspondence[id1 * 2 + 1] = best_y2;
        
        // Copy match data
        for (int c = 0; c < c1; c++) {
            int src_idx = c * size2 + (best_y2 * w2 + best_x2);
            int dest_idx = c * size1 + id1;
            match[dest_idx] = target[src_idx];
        }
    }
}