#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void hist_remap2_kernel( float *I, int nI, float *mI, float *histJ, float *cumJ, float *_minJ, float *_maxJ, int nbins, float *_sortI, int *_idxI, float *R, int c, int h, int w )
{
    int _id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;

    if (_id >= c * size) return; // Early exit if outside bounds

    int id = _id % size, dc = _id / size;

    float minJ  = _minJ[dc];
    float maxJ  = _maxJ[dc];
    float stepJ = (maxJ - minJ) / nbins;

    int idxI = _idxI[_id] - 1;
    if (mI[idxI] < EPS) return;

    int offset = h * w - nI;
    int cdf = id - offset;

    int s = 0;
    int e = nbins - 1;
    while (s <= e) {
        int m = (s + e) / 2; // Move m inside the loop
        float cdf_e = m == nbins - 1 ? cumJ[dc * nbins + m] + 0.5f : cumJ[dc * nbins + m];
        float cdf_s = m == 0 ? -0.5f : cumJ[dc * nbins + m - 1];

        if (cdf >= cdf_e) {
            s = m + 1;
        } else if (cdf < cdf_s) {
            e = m - 1;
        } else {
            float hist  = histJ[dc * nbins + m];
            float ratio = MIN(MAX((cdf - cdf_s) / (hist + 1e-8), 0.0f), 1.0f);
            float activation = minJ + (static_cast<float>(m) + ratio) * stepJ;
            R[dc * size + idxI] = activation;
            break;
        }
    }
}