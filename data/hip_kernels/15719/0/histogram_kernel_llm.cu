#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void histogram_kernel(float *I, float *minI, float *maxI, float *mask, int nbins, int c, int h, int w, float *hist) {
    int _id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;

    // Ensure the thread processes valid data
    if (_id >= c * size) return;
    
    int id = _id % size, dc = _id / size;

    // Check mask threshold
    if (mask[id] < EPS) return;

    float val = I[_id];
    float _minI = minI[dc];
    float _maxI = maxI[dc];

    // Adjust identical min and max
    if (_minI == _maxI) {
        _minI -= 1.0f;
        _maxI += 1.0f;
    }

    // Valid value range check 
    if (_minI <= val && val <= _maxI) {
        // Calculate bin index safely without intermediate overflow
        int idx = __float2int_rz((val - _minI) * nbins / (_maxI - _minI));
        idx = MIN(idx, nbins - 1);
        int index = dc * nbins + idx;

        // Use atomic operation for histogram update
        atomicAdd(&hist[index], 1.0f);
    }
}