#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void upsample_corr_kernel(int *curr_corr, int *next_corr, int curr_h, int curr_w, int next_h, int next_w)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < next_h * next_w) {
        int next_x = id % next_w, next_y = id / next_w;

        // Precompute reciprocal ratios for efficiency.
        float inv_w_ratio = (float)curr_w / next_w;
        float inv_h_ratio = (float)curr_h / next_h;

        int curr_x = (next_x + 0.5) * inv_w_ratio;
        int curr_y = (next_y + 0.5) * inv_h_ratio;

        curr_x = MAX(MIN(curr_x, curr_w-1), 0);
        curr_y = MAX(MIN(curr_y, curr_h-1), 0);

        int curr_id = curr_y * curr_w + curr_x;

        int curr_x2 = curr_corr[2 * curr_id + 0];
        int curr_y2 = curr_corr[2 * curr_id + 1];

        int next_x2 = next_x + (curr_x2 - curr_x) / inv_w_ratio + 0.5;
        int next_y2 = next_y + (curr_y2 - curr_y) / inv_h_ratio + 0.5;

        next_x2 = MAX(MIN(next_x2, next_w-1), 0);
        next_y2 = MAX(MIN(next_y2, next_h-1), 0);

        next_corr[2 * id + 0] = next_x2;
        next_corr[2 * id + 1] = next_y2;
    }
}