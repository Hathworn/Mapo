#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define PIXELS 3073
#define HIDDEN_LAYER_1 2000
#define HIDDEN_LAYER_2 450
#define OUTPUT_LAYER 10
#define LEARNING_RATE 0.01
#define ELEMENTS 1000
#define BLOCKS 32

__global__ void get_layer(double *input, double *matrix, double *result, int input_size, int hidden_size) {
    // Compute unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index is within bounds
    if (idx < hidden_size) {
        double sum = 0.0;  // Initialize sum for reduction

        // Perform dot product for each hidden layer output element
        for (int j = 0; j < input_size; ++j) {
            sum += input[j] * matrix[j * hidden_size + idx];
        }

        // Store the result
        result[idx] = sum;
    }
}