#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double2 complex_multiply(double2 z1, double2 z2)
{
    double real = z1.x * z2.x - z1.y * z2.y;
    double imag = z1.y * z2.x + z1.x * z2.y;
    return make_double2(real, imag);
}

__global__ void inverse_dft_with_w_correction(double2 *grid, size_t grid_pitch, const double3 *visibilities, const double2 *vis_intensity, int vis_count, int batch_count, int x_offset, int y_offset, int render_size, double cell_size)
{
    // Compute global thread indices
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Return if out of bounds
    if (idx >= render_size || idy >= render_size) return;

    double real_sum = 0.0;
    double imag_sum = 0.0;

    // Calculate x and y image coordinates
    double x = (idx + x_offset) * cell_size;
    double y = (idy + y_offset) * cell_size;

    double2 theta_complex;
    double image_correction = sqrt(1.0 - (x * x) - (y * y));
    double w_correction = image_correction - 1.0;

    // Loop through all visibilities
    for (int i = 0; i < batch_count; ++i)
    {
        double theta = 2.0 * M_PI * (x * visibilities[i].x + y * visibilities[i].y + (w_correction * visibilities[i].z));
        double sin_theta, cos_theta;
        sincos(theta, &sin_theta, &cos_theta);
        theta_complex = make_double2(cos_theta, sin_theta);

        double2 vis = complex_multiply(vis_intensity[i], theta_complex);
        real_sum += vis.x;
        imag_sum += vis.y;
    }

    // Apply image correction
    real_sum *= image_correction;
    imag_sum *= image_correction;

    // Write result to grid and normalize by vis_count
    double2 *row = (double2*)((char*)grid + idy * grid_pitch);
    row[idx].x += (real_sum / vis_count);
    row[idx].y += (imag_sum / vis_count);
}