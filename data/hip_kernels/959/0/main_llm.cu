#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "inverse_dft_with_w_correction_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    double2 *grid = NULL;
hipMalloc(&grid, XSIZE*YSIZE);
size_t grid_pitch = 2;
const double3 *visibilities = NULL;
hipMalloc(&visibilities, XSIZE*YSIZE);
const double2 *vis_intensity = NULL;
hipMalloc(&vis_intensity, XSIZE*YSIZE);
int vis_count = 1;
int batch_count = 2;
int x_offset = 1;
int y_offset = 1;
int render_size = XSIZE*YSIZE;
double cell_size = XSIZE*YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
inverse_dft_with_w_correction<<<gridBlock, threadBlock>>>(grid,grid_pitch,visibilities,vis_intensity,vis_count,batch_count,x_offset,y_offset,render_size,cell_size);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
inverse_dft_with_w_correction<<<gridBlock, threadBlock>>>(grid,grid_pitch,visibilities,vis_intensity,vis_count,batch_count,x_offset,y_offset,render_size,cell_size);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
inverse_dft_with_w_correction<<<gridBlock, threadBlock>>>(grid,grid_pitch,visibilities,vis_intensity,vis_count,batch_count,x_offset,y_offset,render_size,cell_size);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}