#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(unsigned char *ptr, int ticks) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // cache constant values and avoid repetitive computations
    float fx = x - DIM / 2;
    float fy = y - DIM / 2;
    float d = sqrtf(fx * fx + fy * fy);
    float cos_val = cosf(d / 10.0f - ticks / 7.0f);
    float denom = d / 10.0f + 1.0f;
    unsigned char grey = (unsigned char)(128.0f + 127.0f * cos_val / denom);

    // write values to memory with minimal computation
    ptr[offset * 4 + 0] = grey;
    ptr[offset * 4 + 1] = grey;
    ptr[offset * 4 + 2] = grey;
    ptr[offset * 4 + 3] = 255;
}