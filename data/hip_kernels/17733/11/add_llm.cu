#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int addem(int a, int b) {
    return a + b;
}

__global__ void add(int a, int b, int *c) {
    // Use a shared variable to store the result to reduce global memory access
    __shared__ int result;
    if (threadIdx.x == 0) {
        result = addem(a, b);
        *c = result;
    }
    __syncthreads();  // Ensure all threads have finished computation
}