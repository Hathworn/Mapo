#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(int *a, int *b, int *c) {
    // Calculate the global index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure index is within bounds
    if (idx < N) {
        // Efficiently calculate indices with modulo operation
        int idx1 = (idx + 1) & 255;  // Use bitwise AND as a faster modulo with 256
        int idx2 = (idx + 2) & 255;

        // Combine calculations to reduce the number of operations
        float sum_a = a[idx] + a[idx1] + a[idx2];
        float sum_b = b[idx] + b[idx1] + b[idx2];

        // Compute the average and assign to c
        c[idx] = (sum_a + sum_b) / 6.0f;  // Dividing by 6 instead of 3 twice
    }
}