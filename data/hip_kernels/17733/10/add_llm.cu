#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel by removing global memory access
__global__ void add(int a, int b, int *c) {
    int result = a + b;  // Perform addition in a register
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *c = result;  // Write result back to global memory only once
    }
}