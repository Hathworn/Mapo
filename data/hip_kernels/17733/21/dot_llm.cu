#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(float *a, float *b, float *c) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0f;
    // Unroll loops for efficiency
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        temp += a[i] * b[i];
    }

    // Save in shared memory
    cache[cacheIndex] = temp;

    // Synchronize threads in this block
    __syncthreads();

    // Perform reduction using shared memory
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    // Store block result
    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}