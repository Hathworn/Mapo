#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add( int *a, int *b, int *c ) {
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a more efficient for loop with stride
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < N; i += stride) {
        c[i] = a[i] + b[i];
    }
}