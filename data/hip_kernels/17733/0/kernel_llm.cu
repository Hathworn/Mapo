#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)

__global__ void kernel(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        // Precompute reused indices
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        
        // Load data from global memory once
        int a0 = a[idx], a1 = a[idx1], a2 = a[idx2];
        int b0 = b[idx], b1 = b[idx1], b2 = b[idx2];

        // Compute averages directly
        float as = (a0 + a1 + a2) / 3.0f;
        float bs = (b0 + b1 + b2) / 3.0f;

        // Store result
        c[idx] = (as + bs) * 0.5f;
    }
}