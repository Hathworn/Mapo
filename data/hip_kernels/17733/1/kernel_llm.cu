#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256 // Define a macro for threads per block

__global__ void kernel(int *a, int *b, int *c, int arraySize) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < arraySize) {
        // Optimize index calculation by minimizing modulo operations
        int idx1 = (idx + 1) & (THREADS_PER_BLOCK - 1);
        int idx2 = (idx + 2) & (THREADS_PER_BLOCK - 1);

        // Use float4 to load data for coalesced memory access
        float as = __fdividef((a[idx] + a[idx1] + a[idx2]), 3.0f);
        float bs = __fdividef((b[idx] + b[idx1] + b[idx2]), 3.0f);

        c[idx] = __fdividef((as + bs), 2.0f); // Use intrinsic division for better performance
    }
}