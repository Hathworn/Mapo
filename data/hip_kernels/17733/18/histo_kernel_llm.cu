#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernel( unsigned char *buffer, long size, unsigned int *histo ) {

    // Use shared memory to cache histogram values
    __shared__ unsigned int temp[256];
    temp[threadIdx.x] = 0; // Initialize shared memory
    __syncthreads();

    // Calculate global index and stride for loop iteration
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    // Loop through buffer with stride and accumulate results into shared memory
    while (i < size) {
        atomicAdd(&temp[buffer[i]], 1);
        i += stride;
    }

    __syncthreads();  // Ensure all writes to shared memory are complete

    // Update global histogram using accumulated results from shared memory
    atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
}