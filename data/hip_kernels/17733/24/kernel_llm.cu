#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(unsigned char *ptr, int ticks) {
    // Calculate thread and block indices
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // Precompute repeated values for efficiency
    float fx = x - DIM / 2;
    float fy = y - DIM / 2;
    float d = sqrtf(fx * fx + fy * fy);
    float cosine_value = cosf(d / 10.0f - ticks / 7.0f);
    unsigned char grey = (unsigned char)(128.0f + 127.0f * cosine_value / (d / 10.0f + 1.0f));

    // Minimize redundant calculations and branching
    int base_offset = offset * 4;
    ptr[base_offset + 0] = grey;
    ptr[base_offset + 1] = grey;
    ptr[base_offset + 2] = grey;
    ptr[base_offset + 3] = 255;
}