#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    // Use both blockIdx and threadIdx to compute unique tid for parallel execution
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Total number of threads

    // Optimize loop to use stride for better GPU utilization
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += stride;
    }
}