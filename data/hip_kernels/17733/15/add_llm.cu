#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    // Optimize: Use blockDim and threadIdx for parallel execution within blocks
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}