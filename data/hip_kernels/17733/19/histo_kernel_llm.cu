#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histo_kernel( unsigned char *buffer, long size, unsigned int *histo ) {
    // Use shared memory for histogram to reduce global memory atomic adds
    __shared__ unsigned int localHisto[256];
    int tid = threadIdx.x;

    // Initialize shared histogram
    if (tid < 256) {
        localHisto[tid] = 0;
    }
    __syncthreads();

    // Calculate the starting index and the stride for each thread
    int i = tid + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Accumulate into shared memory histogram
    while (i < size) {
        atomicAdd(&localHisto[buffer[i]], 1);
        i += stride;
    }
    __syncthreads();

    // Aggregate shared memory histogram into global memory
    if (tid < 256) {
        atomicAdd(&histo[tid], localHisto[tid]);
    }
}