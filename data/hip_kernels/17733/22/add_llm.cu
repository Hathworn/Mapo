#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (tid < n) // Use global index for boundary check
        c[tid] = a[tid] + b[tid];
}