#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel()
{
    // Use constants to reduce calculations
    const int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    const int tidy = threadIdx.y + blockIdx.y * blockDim.y;

    // Pre-calculate message size
    const int size = sizeof(int);

    // Print debugging information
    printf("Hello, World! size=%d   tidx=%d, tidy=%d\n", size, tidx, tidy);
}