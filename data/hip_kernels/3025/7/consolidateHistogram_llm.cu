#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void consolidateHistogram(ulong* blockHistograms, ulong* cudaHistogram, uint numBlocks) {
    int tid = threadIdx.x;
    // Use a shared memory to reduce global memory operations
    __shared__ ulong localSum[256];
    localSum[tid] = 0;

    // Unroll the loop for better performance
    for (uint j = tid; j < numBlocks * 256; j += 256) {
        localSum[tid] += blockHistograms[j];
    }
    __syncthreads();

    // Reduce results within the block
    for (uint offset = 128; offset > 0; offset >>= 1) {
        if (tid < offset) {
            localSum[tid] += localSum[tid + offset];
        }
        __syncthreads();
    }

    // Accumulate the result back to global memory for the first thread in the block
    if (tid == 0) {
        atomicAdd(&cudaHistogram[blockIdx.x], localSum[0]);
    }
}