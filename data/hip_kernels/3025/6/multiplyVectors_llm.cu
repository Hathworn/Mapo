#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiplyVectors(float* A, float* B, float* C, int WIDTH, int HEIGHT) {
    // Calculate global thread ID for column and row
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure within boundaries
    if (x < WIDTH && y < HEIGHT) {
        double result = 0.0;
        
        // Use shared memory to optimize memory access patterns
        extern __shared__ float sharedB[];
        
        for (int i = 0; i < WIDTH; i += blockDim.x) {
            // Load a block of B columns into shared memory
            if (i + threadIdx.x < WIDTH)
                sharedB[threadIdx.y * blockDim.x + threadIdx.x] = B[(i + threadIdx.x) * WIDTH + x];
            __syncthreads();
            
            // Perform multiplication using the loaded block of B
            for (int j = 0; j < blockDim.x && i + j < WIDTH; ++j) {
                result += A[y * WIDTH + (i + j)] * sharedB[threadIdx.y * blockDim.x + j];
            }
            __syncthreads();
        }

        C[y * WIDTH + x] = result;
    }
}