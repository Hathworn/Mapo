#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void test(int* input, int* output) {
    // Calculate global thread index within grid
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    int index = tx + ty * gridDim.x * blockDim.x;  // Enhance memory coalescing
    output[index] = input[index] * 2;
}