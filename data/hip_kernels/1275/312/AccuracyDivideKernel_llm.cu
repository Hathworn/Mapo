#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Optimization: Use blockIdx.x and threadIdx.x to ensure single-thread execution
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        *accuracy /= N;
    }
}