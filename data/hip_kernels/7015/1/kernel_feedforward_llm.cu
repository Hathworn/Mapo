#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

namespace ann {








// CUDA2





}

__global__ void kernel_feedforward( int layer_id, int *l, int *s, int *sw, float *z_arr, float *a_arr, float *w_arr ){
    // Get the global thread index
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    // Get neuron counts for the current and previous layers
    int neuron_count = l[layer_id];
    int neuron_count_prev = l[layer_id-1];

    // Return if index is out of bounds
    if(idx >= neuron_count-1) return;

    // Initialize z accumulator
    float z = 0;

    // Optimize by unrolling loop manually with pragma, reducing loop overhead
    #pragma unroll
    for(int k = 0; k < neuron_count_prev; k++){
        // Accumulate weighted input
        z += w_arr[sw[layer_id-1] + k*(neuron_count - 1) + idx] * a_arr[s[layer_id-1] + k];
    }

    // Compute and store the weighted sum
    z_arr[s[layer_id] + idx] = z;

    // Apply activation function and store output
    float a = 1.0f / (1.0f + expf(-z));
    a_arr[s[layer_id] + idx] = a;
}