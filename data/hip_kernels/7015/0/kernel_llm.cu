#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

namespace ann {








// CUDA2





}

__global__ void kernel(int n, float *arr){

    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if(idx < n) { // Process only valid indices
        float val = arr[idx];
        val *= 2.0f;
        arr[idx] = val;
    }
}