#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

namespace ann {

}

__global__ void kernel_weight_update_2(int layer_id, int *l, int *s_ext, int *sw_ext, float *z_ext_arr, float *a_ext_arr, float *t_arr, float *gjl_ext, float *w_ext_arr, float *dw_ext_arr, float eta, float alpha) {

    // Calculate thread indices and check bounds
    int idx = threadIdx.y + blockDim.y * blockIdx.y;
    if (idx >= l[layer_id]) return;

    int pidx = threadIdx.x;
    int h = blockDim.x;
    int neuron_count_next = l[layer_id + 1];

    float a = a_ext_arr[s_ext[layer_id] + idx];

    int index0 = s_ext[layer_id + 1] + pidx;
    int index1 = sw_ext[layer_id] + idx * (neuron_count_next - 1) + pidx;

    // Unroll loop to improve performance
    for (int k = pidx; k < neuron_count_next - 1; k += h) {
        // Calculate gradient and weight update
        float grad = a * gjl_ext[index0];
        float dw = -eta * grad + alpha * dw_ext_arr[index1];
        
        // Update weights
        dw_ext_arr[index1] = dw;
        w_ext_arr[index1] += dw;
        
        index0 += h;
        index1 += h;
    }
}