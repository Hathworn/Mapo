#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

namespace ann {

}

__global__ void kernel_calc_gL(int layer_id, int *l, int *s, float *z_arr, float *a_arr, float *t_arr, float *gjl) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx >= l[layer_id]) return; // Check neuron count directly

    int neuron_idx = s[layer_id] + idx;
    float z = z_arr[neuron_idx];
    float tmp = 1 + expf(-z);
    float f_deriv = expf(-z) / (tmp * tmp);

    gjl[neuron_idx] = f_deriv * (a_arr[neuron_idx] - t_arr[idx]);
}