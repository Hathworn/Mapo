#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

namespace ann {

}

__global__ void kernel_calc_gL_2(int layer_id, int *l, int *s_ext, float *z_ext_arr, float *a_ext_arr, float *t_arr, float *gjl_ext) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    int neuron_count = l[layer_id];

    if (idx >= neuron_count - 1) return;

    float z = z_ext_arr[s_ext[layer_id] + idx];
    
    // Calculate exp(-z) once and reuse
    float exp_neg_z = expf(-z);
    
    // Use the common subexpression to avoid redundant computation
    float tmp = 1 + exp_neg_z;
    float f_deriv = exp_neg_z / (tmp * tmp);

    // Store result directly
    gjl_ext[s_ext[layer_id] + idx] = f_deriv * (a_ext_arr[s_ext[layer_id] + idx] - t_arr[idx]);
}