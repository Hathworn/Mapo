#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

namespace ann {








// CUDA2





}

__global__ void kernel_calc_gjL( int layer_id, int *l, int *s, int *sw, float *z_arr, float *a_arr, float *t_arr, float *gjl, float *w_arr ){

    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    int neuron_count = l[layer_id];
    int neuron_count_next = l[layer_id + 1];

    if (idx >= neuron_count - 1) return;

    float z = z_arr[s[layer_id] + idx];
    // Precompute expf(-z) for efficiency
    float exp_neg_z = expf(-z);  
    float tmp = 1 + exp_neg_z;
    // Derivative calculation optimized by reusing exp_neg_z
    float f_deriv = exp_neg_z / (tmp * tmp);

    float sum = 0;
    int offset_next_layer = sw[layer_id] + idx * (neuron_count_next - 1);
    int gjl_offset_next_layer = s[layer_id + 1];
    // Unroll loop for potential better performance (compiler might optimize further)
    for (int k = 0; k < neuron_count_next - 1; k++) {
        sum += w_arr[offset_next_layer + k] * gjl[gjl_offset_next_layer + k];
    }

    gjl[s[layer_id] + idx] = f_deriv * sum;
    // printf("Kernelis %d - %.20f\n", s[layer_id] + idx, gjl[s[layer_id] + idx]);
}