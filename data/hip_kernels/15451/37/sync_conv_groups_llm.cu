#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void sync_conv_groups() {
    // Using __syncthreads() to ensure all threads synchronize
    __syncthreads();  
}