#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(float *a, float *b, float *c)
{
    // Use thread ID to handle multiple elements
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    c[idx] = a[idx] + b[idx];
}