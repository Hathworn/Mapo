#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//!!nvcc -c  test.cu --compiler-options -fPIC
//!g++ -o program -L/usr/local/cuda/lib64 main.cpp test.o -lcuda -lcudart

__global__ void mul(float *a, float *b, float *c)
{
    // Use thread ID as an index for parallel operations
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Perform multiplication using indexed access
    c[idx] = a[idx] * b[idx];
}