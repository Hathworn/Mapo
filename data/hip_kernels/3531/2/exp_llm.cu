#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel to use multi-threading for batched computation
__global__ void exp(float *a, float *c, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        c[idx] = expf(a[idx]);  // Use parallel computation for array
    }
}