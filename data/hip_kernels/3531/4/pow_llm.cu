#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized pow kernel function
__global__ void pow(float *a, float *b, float *c)
{
    // Using shared memory to reduce global memory access
    __shared__ float shared_a;
    __shared__ float shared_b;

    // Load data into shared memory
    shared_a = *a;
    shared_b = *b;

    // Ensure all threads have loaded shared memory
    __syncthreads();

    // Compute power using shared memory values
    *c = powf(shared_a, shared_b);
}