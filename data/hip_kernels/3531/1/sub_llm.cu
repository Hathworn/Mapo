#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize by using threadIdx and blockIdx for parallel execution
__global__ void sub(float *a, float *b, float *c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Ensure idx does not exceed array bounds
    if (idx < 1) {
        c[idx] = a[idx] - b[idx];
    }
}