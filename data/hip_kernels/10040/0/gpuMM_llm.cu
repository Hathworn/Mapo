#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpuMM(float *A, float *B, float *C, int N)
{
    // Matrix multiplication for NxN matrices C=A*B
    // Optimize thread computation using shared memory
    __shared__ float sharedA[TILE_SIZE][TILE_SIZE];
    __shared__ float sharedB[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.f;

    for (int i = 0; i < N; i += TILE_SIZE)
    {
        // Load data into shared memory
        sharedA[threadIdx.y][threadIdx.x] = A[row * N + i + threadIdx.x];
        sharedB[threadIdx.y][threadIdx.x] = B[(i + threadIdx.y) * N + col];
        
        __syncthreads(); // Synchronize threads
        
        // Perform multiplication
        for (int k = 0; k < TILE_SIZE; ++k)
        {
            sum += sharedA[threadIdx.y][k] * sharedB[k][threadIdx.x];
        }

        __syncthreads(); // Synchronize threads
    }

    C[row * N + col] = sum;
}