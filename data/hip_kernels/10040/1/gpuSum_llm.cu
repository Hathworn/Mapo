#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpuSum(int *prices, int *sumpricesout, int days, int seconds, int N) 
{
    // Use shared memory for better memory access efficiency
    extern __shared__ int shared_prices[];
    int currentday = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (currentday < days) 
    {
        int start = currentday * seconds;
        int end = start + seconds;

        // Load data into shared memory
        int local_totprice = 0;
        for (int j = threadIdx.x; j < seconds; j += blockDim.x) 
        {
            int index = start + j;
            if (index < end) 
            {
                shared_prices[j] = prices[index];
            }
        }
        
        __syncthreads(); // Synchronize threads after loading data to shared memory

        // Compute local total price using shared memory
        for (int j = threadIdx.x; j < seconds; j += blockDim.x) 
        {
            local_totprice += shared_prices[j];
        }

        // Use atomic addition to update the result
        atomicAdd(&sumpricesout[currentday], local_totprice);
    }
}