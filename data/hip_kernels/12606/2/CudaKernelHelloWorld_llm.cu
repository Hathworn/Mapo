#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CudaKernelHelloWorld(char *a, int *b)
{
    // Using blockDim.x for global indexing to support more threads
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    a[idx] += b[idx];
}