#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define PI 3.1415926536f

// Kernel function optimized for better readability and performance

extern "C"
__global__ void CudaKernelTextureCubeStrobelight(char *surface, int width, int height, size_t pitch, int face, float t)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Early return for threads not corresponding to valid pixels
    if (x >= width || y >= height) return;

    // Optimize variable access and computation
    unsigned char* pixel = (unsigned char*)(surface + y * pitch) + 4 * x;

    // Precompute common expressions
    float theta_x = (2.0f * x) / width - 1.0f;
    float theta_y = (2.0f * y) / height - 1.0f;
    float theta = 2.0f * PI * sqrtf(theta_x * theta_x + theta_y * theta_y);
    
    // Use improved cosine computation
    unsigned char value = static_cast<unsigned char>(255.0f * (0.6f + 0.4f * cosf(theta + t)));
    
    // Set alpha channel once for clarity and performance
    pixel[3] = 255; 

    // Use ternary operator for concise conditional assignment
    pixel[0] = (face % 2) ? pixel[face / 2] : value;  
    pixel[1] = (face % 2) ? pixel[face / 2] : value;  
    pixel[2] = (face % 2) ? 0.5f : value;  
}