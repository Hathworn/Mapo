#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define threshold 1e-4
#define n (2048)
void init(void);
void ref(void);
void test(void);
void compare(int N, double *wref, double *w);

__global__ void test_kernel(int N, double *A, double *B, double *X)
{
    // Each thread takes care of one row of X
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if(k < n) {
        for (int i = 0; i < n; i++) {
            double temp = B[i * N + k]; // Fetch B element
            for (int j = 0; j < i; j++) {
                // Accumulate subtraction for each prior element
                temp -= A[j * N + i] * X[j * N + k];
            }
            X[i * N + k] = temp / A[i * N + i]; // Compute result for X
        }
    }
}