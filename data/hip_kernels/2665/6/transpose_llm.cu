#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose(int N, double *A)
{
    // Calculate row and column indices using 2D mapping for better readability
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure indices are within matrix bounds and reduce unnecessary swaps
    if (row < N && col < N && row < col) {
        double temp = A[row * N + col];
        A[row * N + col] = A[col * N + row];
        A[col * N + row] = temp;
    }
}