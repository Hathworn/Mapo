#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* inference-101
*/

#define COLOR_COMPONENT_MASK            0x3FF
#define COLOR_COMPONENT_BIT_SIZE        10

#define FIXED_DECIMAL_POINT             24
#define FIXED_POINT_MULTIPLIER          1.0f
#define FIXED_COLOR_COMPONENT_MASK      0xffffffff

#define MUL(x,y)    (x*y)

__constant__ uint32_t constAlpha;
__constant__ float  constHueColorSpaceMat[9];

__device__ void YUV2RGB(uint32_t *yuvi, float *red, float *green, float *blue)
{
    const float luma = float(yuvi[0]);
    const float u    = float(yuvi[1]) - 512.0f;
    const float v    = float(yuvi[2]) - 512.0f;

    *red    = luma + 1.140f * v;
    *green  = luma - 0.395f * u - 0.581f * v;
    *blue   = luma + 2.032f * u;
}

__global__ void NV12ToRGBAf(uint32_t* srcImage, size_t nSourcePitch, float4* dstImage, size_t nDestPitch, uint32_t width, uint32_t height)
{
    int x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return; // Early exit for out-of-bounds threads

    uint32_t yuv101010Pel[2];
    uint32_t processingPitch = nSourcePitch;
    uint8_t *srcImageU8 = (uint8_t *)srcImage;

    // Read luma components
    yuv101010Pel[0] = (srcImageU8[y * processingPitch + x]) << 2;
    yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]) << 2;

    uint32_t chromaOffset = processingPitch * height;
    int y_chroma = y >> 1;

    uint32_t chromaCb = srcImageU8[chromaOffset + y_chroma * processingPitch + x];
    uint32_t chromaCr = srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1];

    // Interpolate chroma if on an odd scanline
    if (y & 1 && y_chroma < ((height >> 1) - 1))
    {
        chromaCb = (chromaCb + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x] + 1) >> 1;
        chromaCr = (chromaCr + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x + 1] + 1) >> 1;
    }

    yuv101010Pel[0] |= (chromaCb << COLOR_COMPONENT_BIT_SIZE + 2);
    yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

    yuv101010Pel[1] |= (chromaCb << COLOR_COMPONENT_BIT_SIZE + 2);
    yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

    uint32_t yuvi[6];
    float red[2], green[2], blue[2];

    yuvi[0] = (yuv101010Pel[0] &   COLOR_COMPONENT_MASK);
    yuvi[1] = ((yuv101010Pel[0] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
    yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    yuvi[3] = (yuv101010Pel[1] &   COLOR_COMPONENT_MASK);
    yuvi[4] = ((yuv101010Pel[1] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK);
    yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    // YUV to RGB Transformation conversion
    YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
    YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

    const float s = 1.0f / 1024.0f * 255.0f;

    dstImage[y * width + x]     = make_float4(red[0] * s, green[0] * s, blue[0] * s, 1.0f);
    dstImage[y * width + x + 1] = make_float4(red[1] * s, green[1] * s, blue[1] * s, 1.0f);
}