#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RGBToRGBAf(uchar3* srcImage, float4* dstImage, uint32_t width, uint32_t height)
{
    // Calculate the x and y coordinates
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Check if the coordinates are within bounds
    if (x < width && y < height) 
    {
        int pixel = y * width + x;

        // Load the pixel and convert to float4
        const uchar3 px = srcImage[pixel];
        dstImage[pixel] = make_float4(px.x, px.y, px.z, 255.0f);
    }
}