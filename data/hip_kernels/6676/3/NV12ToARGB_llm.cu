#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* inference-101
*/

#define COLOR_COMPONENT_MASK            0x3FF
#define COLOR_COMPONENT_BIT_SIZE        10

#define FIXED_DECIMAL_POINT             24
#define FIXED_POINT_MULTIPLIER          1.0f
#define FIXED_COLOR_COMPONENT_MASK      0xffffffff

#define MUL(x,y)    (x*y)

__constant__ uint32_t constAlpha;
__constant__ float  constHueColorSpaceMat[9];

__device__ uint32_t RGBAPACK_10bit(float red, float green, float blue, uint32_t alpha)
{
    uint32_t ARGBpixel = 0;

    // Clamp final 10 bit results
    red   = min(max(red,   0.0f), 1023.f);
    green = min(max(green, 0.0f), 1023.f);
    blue  = min(max(blue,  0.0f), 1023.f);

    // Convert to 8 bit unsigned integers per color component
    ARGBpixel = ((((uint32_t)red   >> 2) << 24) |
                 (((uint32_t)green >> 2) << 16) |
                 (((uint32_t)blue  >> 2) <<  8) | (uint32_t)alpha);

    return ARGBpixel;
}

__device__ void YUV2RGB(uint32_t *yuvi, float *red, float *green, float *blue)
{
    const float luma = float(yuvi[0]);
    const float u    = float(yuvi[1]) - 512.0f;
    const float v    = float(yuvi[2]) - 512.0f;

    // Perform fast conversion from YUV to RGB
    *red   = luma + 1.140f * v;
    *green = luma - 0.395f * u - 0.581f * v;
    *blue  = luma + 2.032f * u;
}

__global__ void NV12ToARGB(uint32_t *srcImage, size_t nSourcePitch, uint32_t *dstImage, size_t nDestPitch, uint32_t width, uint32_t height)
{
    uint32_t yuv101010Pel[2];
    uint32_t processingPitch = ((width) + 63) & ~63;
    uint32_t dstImagePitch = nDestPitch >> 2;
    uint8_t *srcImageU8 = (uint8_t *)srcImage;

    processingPitch = nSourcePitch;

    int x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds before processing
    if (x >= width || y >= height) return;

    // Read 2 Luma components at a time
    yuv101010Pel[0] = (srcImageU8[y * processingPitch + x]) << 2;
    yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]) << 2;

    uint32_t chromaOffset = processingPitch * height;
    int y_chroma = y >> 1;

    uint32_t chromaCb, chromaCr;
    chromaCb = srcImageU8[chromaOffset + y_chroma * processingPitch + x];
    chromaCr = srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1];

    // Interpolate chroma vertically
    if (y & 1) 
    {
        if (y_chroma < ((height >> 1) - 1)) 
        {
            chromaCb = (chromaCb + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x] + 1) >> 1;
            chromaCr = (chromaCr + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x + 1] + 1) >> 1;
        }
    }
    yuv101010Pel[0] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
    yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

    yuv101010Pel[1] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
    yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

    // Perform the color conversion with optimized operations
    uint32_t yuvi[6];
    float red[2], green[2], blue[2];

    yuvi[0] = (yuv101010Pel[0] & COLOR_COMPONENT_MASK);
    yuvi[1] = ((yuv101010Pel[0] >> COLOR_COMPONENT_BIT_SIZE) & COLOR_COMPONENT_MASK);
    yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    yuvi[3] = (yuv101010Pel[1] & COLOR_COMPONENT_MASK);
    yuvi[4] = ((yuv101010Pel[1] >> COLOR_COMPONENT_BIT_SIZE) & COLOR_COMPONENT_MASK);
    yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
    YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

    // Pack results into destination image
    dstImage[y * dstImagePitch + x] = RGBAPACK_10bit(red[0], green[0], blue[0], constAlpha);
    dstImage[y * dstImagePitch + x + 1] = RGBAPACK_10bit(red[1], green[1], blue[1], constAlpha);
}