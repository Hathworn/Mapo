#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* inference-101
*/

#define COLOR_COMPONENT_MASK            0x3FF
#define COLOR_COMPONENT_BIT_SIZE        10

#define FIXED_DECIMAL_POINT             24
#define FIXED_POINT_MULTIPLIER          1.0f
#define FIXED_COLOR_COMPONENT_MASK      0xffffffff

#define MUL(x,y)    (x*y)

__constant__ uint32_t constAlpha;
__constant__ float  constHueColorSpaceMat[9];

__device__ uint32_t RGBAPACK_8bit(float red, float green, float blue, uint32_t alpha)
{
    uint32_t ARGBpixel = 0;

    // Clamp final 10 bit results
    red   = min(max(red,   0.0f), 255.0f);
    green = min(max(green, 0.0f), 255.0f);
    blue  = min(max(blue,  0.0f), 255.0f);

    // Convert to 8 bit unsigned integers per color component
    ARGBpixel = ((((uint32_t)red)   << 24) |
                (((uint32_t)green) << 16) |
                (((uint32_t)blue)  <<  8) | (uint32_t)alpha);

    return ARGBpixel;
}

__global__ void Passthru(uint32_t *srcImage, size_t nSourcePitch, uint32_t *dstImage, size_t nDestPitch, uint32_t width, uint32_t height)
{
    int x, y;
    uint32_t yuv101010Pel[2];
    uint32_t dstImagePitch = nDestPitch >> 2;
    uint8_t *srcImageU8 = (uint8_t *)srcImage;

    // Calculate x and y coordinates
    x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    y = blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary check
    if (x >= width || y >= height) return;

    // Read 2 Luma components at a time
    yuv101010Pel[0] = srcImageU8[y * nSourcePitch + x];
    yuv101010Pel[1] = srcImageU8[y * nSourcePitch + x + 1];

    // Luma extraction
    float luma[2];
    luma[0] = (float)(yuv101010Pel[0] & 0x00FF);
    luma[1] = (float)(yuv101010Pel[1] & 0x00FF);

    // Output packed RGBA
    dstImage[y * dstImagePitch + x] = RGBAPACK_8bit(luma[0], luma[0], luma[0], constAlpha);
    dstImage[y * dstImagePitch + x + 1] = RGBAPACK_8bit(luma[1], luma[1], luma[1], constAlpha);
}