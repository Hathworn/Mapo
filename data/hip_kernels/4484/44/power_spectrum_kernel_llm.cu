#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;

    // Loop unrolling to enhance performance
    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {
        if (idx > 0) { // Skip special case more efficiently
            float2 val = reinterpret_cast<float2 *>(Ar)[idx];
            float ret = val.x * val.x + val.y * val.y;
            Aw[idx] = use_power ? ret : sqrtf(ret);
        }
    }

    // Optimize special case handling
    if (thread_id == 0) {
        float real = Ar[0];
        float im = Ar[row_length];
        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}