#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Calculate global linear index for individual threads
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure indices are within bounds of the matrix
    if (i < num_rows && j < num_cols) {
        float f = feats[i * ldf + j];
        feats_sq[i * lds + j] = f * f;
    }
}