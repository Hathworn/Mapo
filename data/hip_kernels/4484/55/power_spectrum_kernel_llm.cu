#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    const float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;

    // Utilize shared memory for improved performance
    __shared__ float2 shared_A[CU1DBLOCK];
    
    for (int idx = thread_id; idx < half_length; idx += CU1DBLOCK) {
        if (idx == 0) continue;

        // Load data into shared memory
        shared_A[thread_id] = reinterpret_cast<const float2 *>(Ar)[idx];
        __syncthreads();

        float ret = shared_A[thread_id].x * shared_A[thread_id].x + shared_A[thread_id].y * shared_A[thread_id].y;
        Aw[idx] = use_power ? ret : sqrtf(ret);
    }

    // Process special case with single thread
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}