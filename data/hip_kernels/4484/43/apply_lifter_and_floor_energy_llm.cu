#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    float *feats = features + frame * ldf;

    // Apply lifter coefficients using shared memory for speed up
    if (cepstral_lifter != 0.0f) {
        __shared__ float shared_lifter_coeffs[CU1DBLOCK];
        int c = thread_id;
        while (c < num_cols) {
            if (thread_id < num_cols && c < CU1DBLOCK)
                shared_lifter_coeffs[thread_id] = lifter_coeffs[thread_id];
            __syncthreads();
            float lift = shared_lifter_coeffs[c % CU1DBLOCK];
            float f = feats[c];
            feats[c] = f * lift;
            c += CU1DBLOCK;
        }
    }

    // Thread 0 for each frame will apply energy
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = logf(energy_floor);
        
        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        feats[0] = energy;
    }
}