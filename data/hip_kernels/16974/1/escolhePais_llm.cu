#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//Bibliotecas Basicas

//Biblioteca Thrust

//Biblioteca cuRAND

//PARAMETROS GLOBAIS
const int QUANT_PAIS_AVALIA = 4;
int POP_TAM = 200;
int N_CIDADES = 20;
int BLOCKSIZE = 1024;
int TOTALTHREADS = 2048;
int N_GERA = 100;
const int MUT = 10;
const int MAX = 19;
const int MIN = 0;
const int ELITE = 2;

/*
* Busca por erros nos processos da gpu
*/
__global__ void escolhePais(unsigned int n, unsigned int np, int *paisAle, double *fitness, int *pool) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Iterate over elements assigned to this thread
    for (int i = index; i < n; i += stride) {
        // Initialize best fitness to a large number
        double best = 10000.0;
        int best_index = -1;

        // Find the best candidate among QUANT_PAIS_AVALIA options
        for (int j = 0; j < QUANT_PAIS_AVALIA; j++) {
            int idx = paisAle[i * QUANT_PAIS_AVALIA + j];
            if (fitness[idx] < best) {
                best = fitness[idx];
                best_index = idx;
            }
        }
        // Store the best index in the pool
        if (best_index != -1) {
            pool[i] = best_index;
        }
    }
}