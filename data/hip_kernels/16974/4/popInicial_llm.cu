#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Global parameters
const int QUANT_PAIS_AVALIA = 4;
int POP_TAM = 200;
int N_CIDADES = 20;
int BLOCKSIZE = 1024;
int TOTALTHREADS = 2048;
int N_GERA = 100;
const int MUT = 10;
const int MAX = 19;
const int MIN = 0;
const int ELITE = 2;

// Kernel function optimized
__global__ void popInicial(unsigned int n, unsigned int np, int* v, int* genes, int* ale)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Unrolling the loop for potential performance improvement
    for (int i = index; i < n; i += stride)
    {
        #pragma unroll
        for(int j = 0; j < np; j++)
        {
            int p = (ale[i * np + j] < j) ? j : ale[i * np + j];
            v[i * np + j] = genes[i * np + p];
            // Swapping with a temporary variable
            int aux = genes[i * np + j];
            genes[i * np + j] = genes[i * np + p];
            genes[i * np + p] = aux;
        }
    }
}