#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel function for setting random numbers

__global__ void setRandom(hiprandState_t* states, int* numbers, int max) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Ensure idx does not exceed array bounds
    if (idx < TOTALTHREADS) {
        numbers[idx] = hiprand(&states[idx]) % max;
    }
}