#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//Bibliotecas Basicas

//Biblioteca Thrust

//Biblioteca cuRAND

//PARAMETROS GLOBAIS
const int QUANT_PAIS_AVALIA = 4;
int POP_TAM = 200;
int N_CIDADES = 20;
int BLOCKSIZE = 1024;
int TOTALTHREADS = 2048;
int N_GERA = 100;
const int MUT = 10;
const int MAX = 19;
const int MIN = 0;
const int ELITE = 2;

/*
* Busca por erros nos processos da gpu
*/
__global__ void cruza(unsigned int n, unsigned int np, int *cidadesAle, int *pop, int *newPop, int *poolPais, int *mutacoes) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Processing in stride to handle larger arrays
    for (int i = index; i < n; i += stride) {
        int paiA = poolPais[i];
        int paiB = poolPais[i + 1];
        int copiaPai = cidadesAle[i * 4];
        int crossover = cidadesAle[(i + 1) * 4] % np;
        int mutar = cidadesAle[(i + 2) * 4];
        int pontoMutar = cidadesAle[(i + 3) * 4] % np;

        // Copy elite parents directly
        if (copiaPai < ELITE) {
            for (int j = 0; j < np; j++) {
                newPop[i * np + j] = pop[paiA * np + j];
            }
            continue;  // Skip crossover and mutation for elite
        }

        // Initial copy from parent A
        for (int j = 0; j < np; j++) {
            newPop[i * np + j] = pop[paiA * np + j];
        }

        // Perform crossover from parent B
        int crossoverEnd = min(crossover + mutacoes[i], MAX);
        for (int j = crossover; j < crossoverEnd; j++) {
            int t = 0;
            while (newPop[i * np + t] != pop[paiB * np + j]) {
                t++;
            }
            // Swap elements
            int aux = newPop[i * np + j];
            newPop[i * np + j] = newPop[i * np + t];
            newPop[i * np + t] = aux;
        }

        // Perform mutation if necessary
        if (mutar < MUT) {
            int mut = clamp(mutacoes[i], MIN, MAX);
            int t = 0;
            while (newPop[i * np + t] != mut) {
                t++;
            }
            // Swap elements for mutation
            int aux = newPop[i * np + pontoMutar];
            newPop[i * np + pontoMutar] = newPop[i * np + t];
            newPop[i * np + t] = aux;
        }
    }
}
