#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 16

__global__ void hello()
{
    // Calculate the global thread index for more unique identification
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello world! I'm a thread %d in block %d, global thread id %d\n", 
           threadIdx.x, blockIdx.x, globalThreadId);
}