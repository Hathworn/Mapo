#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_random_states(unsigned int seed, hiprandState_t* states, size_t num_states)
{
int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
if(thread_id > num_states)
return;

hiprand_init(seed, thread_id, 0, &states[thread_id]);
}