#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_random_states(unsigned int seed, hiprandState_t* states, size_t num_states)
{
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use < instead of > to correctly handle the range check
    if (thread_id >= num_states)
        return;

    // Initialize the random state for this thread
    hiprand_init(seed, thread_id, 0, &states[thread_id]);
}