#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use atomic operation to prevent race conditions if multiple threads are used
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}