#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

}

__global__ void vdiv(const float *a, const float *b, float *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Only perform division if index is within bounds
    if (i < n) {
        c[i] = a[i] / b[i];
    }
}