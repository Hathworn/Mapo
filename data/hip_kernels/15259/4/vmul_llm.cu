#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

}

__global__ void vmul(const float *a, const float *b, float *c, const int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if (i < n) {
        c[i] = a[i] * b[i];
    }
}