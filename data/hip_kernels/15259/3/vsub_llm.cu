#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

}

__global__ void vsub(const float *a, const float *b, float *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index does not exceed the array bounds
    if (i < n) {
        c[i] = a[i] - b[i];
    }
}