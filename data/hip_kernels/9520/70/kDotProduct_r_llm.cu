#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;

    // Loop over elements; take advantage of greater warp efficiency
    while (eidx < numElements) {
        shmem[threadIdx.x] += a[eidx] * b[eidx];
        eidx += gridDim.x * DP_BLOCKSIZE;
    }

    __syncthreads();

    // Unroll reduction within block
    if (threadIdx.x < 256) shmem[threadIdx.x] += shmem[threadIdx.x + 256];
    __syncthreads();
    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    __syncthreads();

    // Use warp shuffle for final reduction
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        float sum = mysh[threadIdx.x];
        sum += __shfl_down_sync(0xffffffff, sum, 16);
        sum += __shfl_down_sync(0xffffffff, sum, 8);
        sum += __shfl_down_sync(0xffffffff, sum, 4);
        sum += __shfl_down_sync(0xffffffff, sum, 2);
        sum += __shfl_down_sync(0xffffffff, sum, 1);
        if (threadIdx.x == 0) target[blockIdx.x] = sum;
    }
}