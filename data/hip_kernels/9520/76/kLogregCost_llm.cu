#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;
    
    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Early assignment of zero in the common case where label probability is not max
        float correctProb = 0.0f;

        // Check if label probability is equal to max probability
        if (labelp == maxp) {
            int numMax = 0;
            // Optimize loop by reducing redundant reads
            float* probPtr = probs + tx;
            for (int i = 0; i < numOut; i++, probPtr += numCases) {
                numMax += (*probPtr == maxp);
            }
            correctProb = 1.0f / float(numMax);
        }
        correctProbs[tx] = correctProb;
    }
}