#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Compute unique global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total number of threads in the grid
    const int numThreads = blockDim.x * gridDim.x;

    // Loop over each element using stride loop pattern for better utilization
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Compute 2D target array coordinates
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        // Map target coordinates to source coordinates
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Copy data from source to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}