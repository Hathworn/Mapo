#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    float tmp = 0.0f;

    // Use warp reduction to optimize the reduction step
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Reduce within a block using warp shuffle
    for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
        tmp += __shfl_down_sync(0xffffffff, tmp, offset);
    }
    
    if (idx == 0) {
        reduction_buffer[0] = tmp;
    }
    __syncthreads();

    // Compute gradient using the reduced result
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}