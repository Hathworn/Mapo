#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate the global thread ID
    const uint tidx = hipGridDim_x * blockIdx.x + threadIdx.x;
    
    // Initialize hiprand with optimized sequence
    hiprand_init(seed, tidx, 0, &state[tidx]);
}