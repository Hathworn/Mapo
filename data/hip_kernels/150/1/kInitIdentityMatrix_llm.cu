#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kInitIdentityMatrix(float* a, int size, int num_elements) {
    // Calculate global index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Stride loop for all elements
    for (int x = idx; x < num_elements; x += blockDim.x * gridDim.x) {
        // Set diagonal elements to 1, others to 0
        a[x] = (x % size == x / size) ? 1.0f : 0.0f;
    }
}