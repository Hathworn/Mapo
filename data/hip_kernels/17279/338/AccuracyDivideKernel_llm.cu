#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel to handle more than one element
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use blockIdx and threadIdx to support parallel execution
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we do not read/write out of bounds
    if (idx < N) {
        accuracy[idx] /= N;
    }
}