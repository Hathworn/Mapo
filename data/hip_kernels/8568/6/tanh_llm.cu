#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tanh(float *inout, float *bias, int rows, int cols) {
    // Use shared memory to minimize global memory access
    extern __shared__ float shared_bias[];
    
    // Load bias into shared memory
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    if (tid < rows) {
        shared_bias[tid] = bias[tid];
    }
    __syncthreads();

    // Optimize index calculation
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (j < cols && i < rows) {
        inout[i * cols + j] = tanhf(inout[i * cols + j]) + shared_bias[i];
    }
}