#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tanh_grad(float *pre_grad, float *output, int rows, int cols) {
    // Compute flattened index from 2D block and thread indices
    int index = blockIdx.y * blockDim.y * cols + blockIdx.x * blockDim.x + threadIdx.y * cols + threadIdx.x;
    
    if (index < rows * cols) { // Ensure the index is within bounds
        float t = output[index];
        pre_grad[index] *= 1 - t * t;
    }
}