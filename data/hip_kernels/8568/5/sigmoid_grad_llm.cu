#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sigmoid_grad(float *pre_grad, float *output, int rows, int cols) {
    // Optimize to use a 1D block and grid structure for better occupancy
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = cols * rows;

    if (idx >= totalThreads) return;

    // Calculate row and column from 1D index
    int i = idx / cols;
    int j = idx % cols;

    float t = output[idx];
    pre_grad[idx] *= t * (1 - t);
}