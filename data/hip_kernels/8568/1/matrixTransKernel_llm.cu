#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
Modified from 
https://github.com/zhxfl/CUDA-CNN
*/

__global__ void matrixTransKernel(float *A, int rows, int cols) {
    // Use shared memory to reduce global memory accesses
    __shared__ float tile[32][33]; // Avoid bank conflicts by using padding
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int width = gridDim.x * blockDim.x;

    // Load matrix into shared memory
    if (x < cols && y < rows) {
        tile[threadIdx.y][threadIdx.x] = A[y * cols + x];
    }
    __syncthreads();

    // Transpose within shared memory and write back to global memory
    x = blockIdx.y * blockDim.y + threadIdx.x;
    y = blockIdx.x * blockDim.x + threadIdx.y;

    if (x < rows && y < cols) {
        A[y * rows + x] = tile[threadIdx.x][threadIdx.y];
    }
}