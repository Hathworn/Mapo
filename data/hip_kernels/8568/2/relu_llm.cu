#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu(float *inout, float *bias, int rows, int cols) {
    // Use shared memory for bias if rows are small; otherwise each thread reads its own
    __shared__ float shared_bias[32];  // Assuming blockDim.y <= 32 for this example
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < cols) {
        if (threadIdx.x == 0) {
            shared_bias[threadIdx.y] = bias[i];
        }
        __syncthreads();  // Ensure shared memory is loaded

        float value = inout[i * cols + j] + shared_bias[threadIdx.y];
        inout[i * cols + j] = fmaxf(0.0, value);
    }
}