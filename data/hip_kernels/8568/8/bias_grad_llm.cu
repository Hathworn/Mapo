#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bias_grad(float *pre_grad, float *output, int rows, int cols) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= rows) return;

    // Initialize output[i] to zero
    float sum = 0;

    // Unroll the loop for better performance
    for (int k = 0; k < cols; k += 4) {
        if (k < cols) sum += pre_grad[i * cols + k];
        if (k + 1 < cols) sum += pre_grad[i * cols + k + 1];
        if (k + 2 < cols) sum += pre_grad[i * cols + k + 2];
        if (k + 3 < cols) sum += pre_grad[i * cols + k + 3];
    }

    // Write the computed sum to output
    output[i] = sum;
}