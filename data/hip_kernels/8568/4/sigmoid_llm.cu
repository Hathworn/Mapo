#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sigmoid(float *inout, float *bias, int rows, int cols) {
    // Calculate global thread index
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Early exit if thread exceeds bounds
    if (j >= cols || i >= rows) return;

    // Efficient memory access and computation
    int index = i * cols + j;
    float t = inout[index];
    inout[index] = 1.0f / (1.0f + expf(-t)) + bias[i];
}