#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
Modified from
https://github.com/zhxfl/CUDA-CNN
*/

__global__ void elementwiseMul(float *x, float *y, float *z, int rows, int cols) {
    // Calculate linear index
    int index = blockIdx.y * blockDim.y * cols + blockIdx.x * blockDim.x + threadIdx.y * cols + threadIdx.x;

    // Check boundary condition
    if (index < rows * cols) {
        // Perform element-wise multiplication
        z[index] = x[index] * y[index];
    }
}