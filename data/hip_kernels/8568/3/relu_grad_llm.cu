#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_grad(float *pre_grad, float *output, int rows, int cols) {
    // Optimize memory access with shared memory
    extern __shared__ float shared_output[];
    
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int index = i * cols + j;

    // Load output into shared memory
    if (j < cols && i < rows) {
        shared_output[threadIdx.y * blockDim.x + threadIdx.x] = output[index];
    }
    __syncthreads();

    // Check and set pre_grad only if within bounds and output is non-positive
    if (j < cols && i < rows && shared_output[threadIdx.y * blockDim.x + threadIdx.x] <= 0) {
        pre_grad[index] = 0;
    }
}