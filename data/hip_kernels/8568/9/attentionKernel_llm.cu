#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void attentionKernel(float *x, int rows, int cols) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (j >= cols) return;
    
    // Use shared memory to store partial sums
    extern __shared__ float sharedSum[];
    sharedSum[threadIdx.x] = 0;

    // Calculate sums using shared memory
    for (int k = 0; k < rows; k++) {
        sharedSum[threadIdx.x] += x[k * cols + j];
    }
    
    // Wait for all threads to calculate their partial sums
    __syncthreads();
    
    // Each thread multiplier by its own sum for simplicity
    float sum = sharedSum[threadIdx.x];
    
    // Multiply using computed sums
    for (int k = 0; k < rows; k++) {
        x[k * cols + j] *= sum;
    }
}