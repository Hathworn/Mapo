#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolution_kernel(unsigned char *input_img, unsigned char *output_img, int height, int width)
{
    __shared__ unsigned char input_shared[W][W]; //Shared Memory required for a tile and its halo elements(3 channels)

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int output_row = blockIdx.x * TILE_WIDTH + tx;
    int output_col = blockIdx.y * TILE_WIDTH + ty;
    int input_row = output_row - MASK_WIDTH / 2;
    int input_col = output_col - MASK_WIDTH / 2;
    int global_idx = (input_row * width + input_col) * 3;

    for (int chan = 0; chan < 3; chan++) // 3 Channel Image
    {
        // Load input data into shared memory with boundary checks
        if (input_row >= 0 && input_row < height && input_col >= 0 && input_col < width) {
            input_shared[tx][ty] = input_img[global_idx + chan];
        } else {
            input_shared[tx][ty] = 0;
        }
        __syncthreads();

        // Only perform convolution on valid output pixels
        if (tx < TILE_WIDTH && ty < TILE_WIDTH && output_row < height && output_col < width) {
            int freq[256] = {0}; // Initialize frequency array

            // Calculate histogram for median filter
            for (int i = 0; i < MASK_WIDTH; i++) {
                for (int j = 0; j < MASK_WIDTH; j++) {
                    freq[input_shared[tx + i][ty + j]]++;
                }
            }

            // Find median
            int median = 0, sum = 0;
            for (int i = 0; i < 256; i++) {
                sum += freq[i];
                if (sum > (MASK_WIDTH * MASK_WIDTH) / 2) {
                    median = i;
                    break;
                }
            }
            
            // Store the median to the output image
            output_img[(output_row * width + output_col) * 3 + chan] = median;
        }
        __syncthreads();
    }
}