#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rgbToGreyKernel(int height, int width, unsigned char *input_img, unsigned char *output_img)
{
    // Combine row and col computation into a single index
    int idx = blockIdx.y * blockDim.y * width + blockIdx.x * blockDim.x + threadIdx.y * width + threadIdx.x;
    
    if (idx < height * width)
    {
        // Reduce repeated index calculations
        int colorIdx = 3 * idx;
        float red = (float)input_img[colorIdx];
        float green = (float)input_img[colorIdx + 1];
        float blue = (float)input_img[colorIdx + 2];

        output_img[idx] = 0.21f * red + 0.71f * green + 0.07f * blue;
    }
}