#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int N, double *a, double *b)
{
    // Use shared memory to minimize repeated access to global memory
    extern __shared__ double shared_a[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < N)
    {
        // Load data into shared memory
        shared_a[threadIdx.x] = a[tid];
        __syncthreads();

        // Perform computation using shared memory
        b[tid] = shared_a[threadIdx.x] * shared_a[threadIdx.x];
    }
}