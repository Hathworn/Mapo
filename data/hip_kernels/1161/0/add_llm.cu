#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int N, double *a, double *b, double *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Unrolling the loop for performance when possible
    int stride = blockDim.x * gridDim.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += stride;
    }
}