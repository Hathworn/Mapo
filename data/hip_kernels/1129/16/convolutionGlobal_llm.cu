#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function
__global__ void convolutionGlobal(float *imgIn, float *kernel, float *imgOut, int w, int h, int nc, int kernelSize) {
    size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    size_t y = threadIdx.y + blockDim.y * blockIdx.y;

    int r = kernelSize / 2;
    
    // Check for boundaries
    if (x >= w || y >= h) return;

    for (unsigned int c = 0; c < nc; c++) {
        float sum = 0;

        // Load x_new and y_new using conditional operators for improved readability
        for (unsigned int i = 0; i < kernelSize; i++) {
            unsigned int x_new = (x + r < i) ? 0 : ((x + r - i >= w) ? w - 1 : x + r - i);

            for (unsigned int j = 0; j < kernelSize; j++) {
                unsigned int y_new = (y + r < j) ? 0 : ((y + r - j >= h) ? h - 1 : y + r - j);

                // Accumulate convolution result
                sum += kernel[i + j * kernelSize] * imgIn[x_new + y_new * w + w * h * c];
            }
        }

        // Store result in the output image
        imgOut[x + y * w + c * w * h] = sum;
    }
}