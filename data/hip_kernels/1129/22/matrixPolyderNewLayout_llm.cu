#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixPolyderNewLayout(const float *coefImg, float *coefImgDer, const int w, const int h, const int m, size_t yOffset) {
    size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    size_t y = threadIdx.y + blockDim.y * blockIdx.y;

    // Early exit if the thread is out of bounds
    if (x >= w || y >= h) return;

    size_t xOffsetDer = m - 1;
    size_t yOffsetDer = w * xOffsetDer;
    size_t xOffsetCoef = m;
    size_t yOffsetCoef = w * xOffsetCoef;

    // Unrolling loop to reduce the overhead and improve performance
    #pragma unroll
    for (int i = 0; i < m - 1; ++i) {
        size_t idxDer = x * xOffsetDer + y * yOffsetDer + i;
        size_t idxCoef = x * xOffsetCoef + y * yOffsetCoef + i;

        // Compute the derivative coefficient
        coefImgDer[idxDer] = coefImg[idxCoef] * (m - i - 1);
    }
}