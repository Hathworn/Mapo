#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void createLaplacianKernel(float *grid, float *kernel, int nrDimensions, int nrGridElements) {
    size_t x = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Ensure x is within bounds
    if (x >= nrGridElements) return;

    // Initialize kernel value with the grid element
    float sum = grid[x];

    // Loop unrolling for better performance
#pragma unroll
    for(int d = 1; d < nrDimensions; ++d) {
        sum += grid[x + d * nrGridElements];
    }

    // Store the computed sum in the kernel
    kernel[x] = sum;
}