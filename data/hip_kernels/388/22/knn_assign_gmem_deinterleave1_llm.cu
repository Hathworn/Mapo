#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel
__global__ void knn_assign_gmem_deinterleave1(uint32_t length, uint16_t k, uint32_t *neighbors) {
    uint64_t sample = blockIdx.x * blockDim.x + threadIdx.x;
    // Early exit if out of range
    if (sample >= length) return;

    // Index calculation for odd and even samples
    uint32_t srcIndex = sample * 2 * k;
    if (sample % 2 == 1) {
        for (int i = 0; i < k; i++) {
            neighbors[sample * k + i] = neighbors[srcIndex + i];
        }
    } else {
        uint32_t destIndex = (length + sample) * k + k;
        for (int i = 0; i < k; i++) {
            neighbors[destIndex + i] = neighbors[srcIndex + i];
        }
    }
}