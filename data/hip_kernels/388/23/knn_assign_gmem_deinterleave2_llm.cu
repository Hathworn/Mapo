#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void knn_assign_gmem_deinterleave2(uint32_t length, uint16_t k, uint32_t *neighbors) {
    uint64_t sample = blockIdx.x * blockDim.x + threadIdx.x;
    sample *= 2;
    
    // Check if the current thread should proceed
    if (sample >= length) {
        return;
    }

    // Compute starting index for source and destination
    uint32_t src_index = (length + sample) * k + k;
    uint32_t dest_index = sample * k;

    // Use loop unrolling to improve performance
    #pragma unroll
    for (int i = 0; i < k; i++) {
        // Copy data from source to destination
        neighbors[dest_index + i] = neighbors[src_index + i];
    }
}