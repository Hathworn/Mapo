#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *r, int *g, int *b, float *gc)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Precompute base index for gc to reduce repetitive calculations
    int baseIdx = 5120 * 6 + i * 6;

    // Use multiplication for bitwise shifts
    gc[baseIdx    ] = b[i] * 0.00390625;
    gc[baseIdx + 1] = g[i] * 0.00390625;
    gc[baseIdx + 2] = r[i] * 0.00390625;

    // Optimize modulus and division with bitwise operations
    gc[baseIdx + 3] = float(i & 511); // Equivalent to: i % 512
    gc[baseIdx + 4] = float(i >> 9);  // Equivalent to: i / 512
    gc[baseIdx + 5] = float(a[i]);
}