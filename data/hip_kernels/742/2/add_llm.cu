#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *r, int *g, int *b, float *gc)
{
    // Calculate the thread's global index once to avoid recalculations
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Use constant float multiplication to convert integers to float values in range
    float factor = 0.00390625;

    // Compute the indices and assign the values with factor multiplication for better code readability and minimal operations
    int index = 5120 * 6 + i * 6;
    gc[index]     = b[i] * factor;
    gc[index + 1] = g[i] * factor;
    gc[index + 2] = r[i] * factor;

    // The below commented operations can be uncommented if needed in the future, place-holding them for potential functionality
    // gc[index + 3] = float(i % 512);
    // gc[index + 4] = float(i / 512);
    // gc[index + 5] = float(a[i]);
}