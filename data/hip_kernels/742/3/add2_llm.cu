#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add2(int *a, int *r, int *g, int *b, float *gc)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Use float conversion directly in the calculation for better precision and performance
    gc[5120 * 6 + i * 6] = b[i] * 0.00390625f;
    gc[5120 * 6 + i * 6 + 1] = g[i] * 0.00390625f;
    gc[5120 * 6 + i * 6 + 2] = r[i] * 0.00390625f;

    // Efficient modulus operation utilizing bitwise and shift operations
    gc[5120 * 6 + i * 6 + 3] = float(i & 511); // i % 512
    gc[5120 * 6 + i * 6 + 4] = float(i >> 9);  // i / 512

    gc[5120 * 6 + i * 6 + 5] = float(a[i]);
}