#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void add(int *a, int *r, int *g, int *b, float *gc)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    int base_index = 5120 * 6 + i * 6;

    // Repeated constant value replaced with formula
    float inv_256 = 0.00390625f;

    // Use precomputed base index and adjusted formulas
    gc[base_index] = b[i] * inv_256;
    gc[base_index + 1] = g[i] * inv_256;
    gc[base_index + 2] = r[i] * inv_256;

    // Simplified modulo and division calculations
    gc[base_index + 3] = float(i & 511);  // i % 512 optimized using bitwise AND
    gc[base_index + 4] = float(i >> 9);   // i / 512 optimized using bitwise shift

    // Direct assignment for last element
    gc[base_index + 5] = float(a[i]);
}