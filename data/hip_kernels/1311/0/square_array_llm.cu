#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square_array(float *a, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        float temp = a[idx];  // Load once
        a[idx] = temp * temp; // Use register variable
    }
}