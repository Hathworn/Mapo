#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void clearLabel(bool *label, unsigned int size)
{
    // Calculate global thread ID
    unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;

    // Use simple conditional to reduce overhead
    if (id < size)
    {
        label[id] = false;
    }
}