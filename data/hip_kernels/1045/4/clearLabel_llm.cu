#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void clearLabel(bool *label, unsigned int size)
{
    // Use the grid stride loop for better scalability across different GPU architectures
    for (unsigned int id = blockIdx.x * blockDim.x + threadIdx.x; id < size; id += blockDim.x * gridDim.x)
    {
        label[id] = false;
    }
}