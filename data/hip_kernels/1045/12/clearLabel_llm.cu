#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void clearLabel(bool *label, unsigned int size)
{
    unsigned int id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID
    if (id < size) {
        label[id] = false; // Set label to false
    }
}