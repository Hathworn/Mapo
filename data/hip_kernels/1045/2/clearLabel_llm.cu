#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void clearLabel(bool *label, unsigned int size)
{
    // Calculate unique thread index
    unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Optimize control flow by using simplified condition
    if(id < size)
    {
        // Clear label using direct memory access
        label[id] = false;
    }
}