#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void clearLabel(bool *label, unsigned int size)
{
    unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Unrolling the loop for better performance
    unsigned int gridSize = gridDim.x * blockDim.x;
    for (unsigned int i = id; i < size; i += gridSize)
    {
        label[i] = false;
    }
}