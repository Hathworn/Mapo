#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void clearLabel(float *prA, float *prB, unsigned int num_nodes, float base)
{
    unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < num_nodes)
    {
        // Use registers for constant values to reduce redundant calculations
        const float factor = 0.85f;
        prA[id] = base + prA[id] * factor;
        prB[id] = 0;
    }
}