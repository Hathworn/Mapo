#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Perform division on the host side before kernel launch for efficiency
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index == 0) { // Ensure that only one thread performs the division
        *accuracy /= N;
    }
}