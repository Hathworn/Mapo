#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

static const int n_el = 512;
static const size_t size = n_el * sizeof(float);

// function which invokes the kernel
__global__ void kernel_sum(const float* A, const float* B, float* C, int n_el)
{
    // Calculate the unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform tid-th elements addition using strided access when needed
    for (int idx = tid; idx < n_el; idx += blockDim.x * gridDim.x) {
        C[idx] = A[idx] + B[idx];
    }
}