#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// This works fine with a mutex, but crashes with a sigbus error when not using a mutex
// #define USE_MUTEX

#ifdef USE_MUTEX
std::mutex m;
#endif


__global__ void testKernel() {
    // Optimized by introducing a sync point to ensure data consistency
    __syncthreads(); // Synchronize threads before printing
    printf("Thread Kernel running\n");
    __syncthreads(); // Synchronize threads after printing
}