#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"


#define _SIZE_ 1000000

/*
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

*/


__global__ void addKernel(int *c, const int *a, const int *b)
{
int i = threadIdx.x;
c[i] = a[i] + b[i];
}