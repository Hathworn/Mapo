#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define _SIZE_ 1000000

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global index to support larger arrays and avoid bank conflicts
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < _SIZE_) { // Ensure index is within bounds
        c[i] = a[i] + b[i];
    }
}