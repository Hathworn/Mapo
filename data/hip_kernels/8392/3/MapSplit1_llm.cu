#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#pragma once

__global__ void MapSplit1(const int* one, int* result, unsigned int mx, unsigned int width)
{
    const unsigned int globalId = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate 2D grid position
    const unsigned int riy = globalId / mx;
    const unsigned int rix = globalId % width;
    
    if (rix < width) { // Ensure valid memory access
        const unsigned int xxx = riy * width + rix;
        const unsigned int ddx = riy * mx + rix;
        result[xxx] = one[ddx];
    }
}