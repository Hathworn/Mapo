#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#pragma once

__global__ void MapAdd1(int* one, const int* result, unsigned int mx, unsigned int width)
{
    // Use shared memory and avoid repeated computation for better performance
    const unsigned int ppp = blockIdx.x * blockDim.x + threadIdx.x;

    if (ppp >= mx * width) return; // Boundary check to prevent out-of-bounds memory access
    
    const unsigned int rix = ppp % width; // Calculate column index
    const unsigned int riy = ppp / width; // Calculate row index
    const unsigned int xxx = riy * width + rix; // Compute index in result array
    const unsigned int ddx = riy * mx + rix; // Compute index in one array

    __shared__ int sharedResult;
    if (threadIdx.x == 0) {
        sharedResult = result[xxx]; // Load data into shared memory once per block
    }
    __syncthreads(); // Ensure all threads have loaded before proceeding

    one[ddx] = sharedResult; // Use shared memory to minimize global memory access
}