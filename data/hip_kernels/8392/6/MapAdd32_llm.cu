#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
#pragma once

__global__ void MapAdd32(int* one, const int* result, unsigned int mx, unsigned int width)
{
    unsigned int ppp = blockIdx.x * blockDim.x * 32 + threadIdx.x;

    #pragma unroll
    for (int i = 0; i < 32; ++i) { // Process 32 elements in a loop
        unsigned int rix = ppp % width;
        unsigned int riy = (ppp / mx) + ((ppp % mx) / width);
        unsigned int xxx = riy * width + rix;
        unsigned int ddx = riy * mx + rix;
        one[ddx] = result[xxx];
        ppp++;
    }
}