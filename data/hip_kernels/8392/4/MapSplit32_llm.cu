#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#pragma once

__global__ void MapSplit32(const int* one, int* result, unsigned int mx, unsigned int width)
{
    unsigned int base_ppp = blockIdx.x * blockDim.x * 32 + threadIdx.x;
    
    // Use loop to process unrolled code parts
    for(unsigned int offset = 0; offset < 32; ++offset) {
        unsigned int ppp = base_ppp + offset;
        unsigned int rix = ppp % width;
        unsigned int riy = (ppp / mx) + ((ppp % mx) / width);
        unsigned int xxx = riy * width + rix;
        unsigned int ddx = riy * mx + rix;
        result[xxx] = one[ddx]; 
    }
}