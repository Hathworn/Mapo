#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy( float *v4, const float *v3, const int n ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate thread global index

    // Use looping instead of manual unrolling of assignment
    for(int i = idx; i < n; i += blockDim.x * gridDim.x) {
        #pragma unroll  // Hint to compiler for loop unrolling
        for (int j = 0; j < 3; ++j) {
            v4[i*8+j] = v3[i*6+j];            // Efficient assignment
        }
        for (int j = 3; j < 6; ++j) {
            v4[i*8+j+1] = v3[i*6+j];          // Efficient assignment with offset
        }
    }
}