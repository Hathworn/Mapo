#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void exclscnmb2e(int *d_data0, int *d_output0, int *d_data1, int *d_output1, int *d_data2, int *d_output2, int *d_data3, int *d_output3, int *d_data4, int *d_output4, int *d_data5, int *d_output5, int *d_data6, int *d_output6, int *d_data7, int *d_output7) {
    const int twid = threadIdx.x;
    int *d_data, *d_output;
    
    // Directly map block index to data and output arrays
    switch (blockIdx.x) {
        case 0: d_data = d_data0; d_output = d_output0; break;
        case 1: d_data = d_data1; d_output = d_output1; break;
        case 2: d_data = d_data2; d_output = d_output2; break;
        case 3: d_data = d_data3; d_output = d_output3; break;
        case 4: d_data = d_data4; d_output = d_output4; break;
        case 5: d_data = d_data5; d_output = d_output5; break;
        case 6: d_data = d_data6; d_output = d_output6; break;
        case 7: d_data = d_data7; d_output = d_output7; break;
        default: return; // Return early if block index is out of bounds
    }

    // Perform computation only if within bounds
    if (twid < 2) {
        d_output[twid] = d_data[0] * twid;
    }
}