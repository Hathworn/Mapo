#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _gather(const float * input, const int * indices, float * output, const int n)
{
    // Use a local variable to avoid repeated calculations
    const int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid < n) {
        // Calculate index once
        int baseIndex = (tid / 6) * 6;
        output[tid] = input[(tid % 6) + baseIndex + indices[tid / 6] * 6];
    }
}