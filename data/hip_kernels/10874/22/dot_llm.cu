#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot( float *a, float *b, float *c ) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0f;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // Store the computed value in cache
    cache[cacheIndex] = temp;

    // Synchronize all threads within the block
    __syncthreads();

    // Perform reduction in shared memory
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads(); // Synchronize after each step
    }

    // Write the result of this block to the output array
    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}