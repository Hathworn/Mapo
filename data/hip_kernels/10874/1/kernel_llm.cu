#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 256

__global__ void kernel(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < N) {
        // Load values from global memory
        int a0 = a[idx];
        int a1 = a[(idx + 1) % 256];
        int a2 = a[(idx + 2) % 256];
        
        int b0 = b[idx];
        int b1 = b[(idx + 1) % 256];
        int b2 = b[(idx + 2) % 256];

        // Compute averages
        float as = (a0 + a1 + a2) / 3.0f;
        float bs = (b0 + b1 + b2) / 3.0f;

        // Store the result
        c[idx] = (as + bs) / 2;
    }
}