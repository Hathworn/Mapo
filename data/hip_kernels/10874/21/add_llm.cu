#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    // Optimize memory access pattern
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Use stride loop to ensure all elements are processed
    for (int i = tid; i < N; i += stride) {
        c[i] = a[i] + b[i];
    }
}