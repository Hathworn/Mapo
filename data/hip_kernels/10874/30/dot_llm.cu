#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define threadsPerBlock 256

__global__ void dot(int size, float *a, float *b, float *c) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;

    // Unroll loop for performance improvement
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        temp += a[i] * b[i];
    }

    // Set the cache values
    cache[cacheIndex] = temp;

    // Synchronize threads in this block
    __syncthreads();

    // Use warp-synchronous reduction for better efficiency
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}