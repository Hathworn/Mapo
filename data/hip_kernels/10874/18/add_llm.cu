#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add( int *a, int *b, int *c ) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // Calculate correct thread id
    int stride = blockDim.x * gridDim.x; // Calculate stride for grid looping
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += stride;
    }
}