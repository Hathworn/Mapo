#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) & 255;  // use bitwise AND for modulo operation
        int idx2 = (idx + 2) & 255;  // use bitwise AND for modulo operation
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = static_cast<int>((as + bs) * 0.5f);  // use static_cast and multiply by 0.5
    }
}