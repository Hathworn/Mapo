#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo) {

    // Declare shared memory as zero-initialized
    __shared__ unsigned int temp[256];

    // Each thread initializes its corresponding location in shared memory
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int bsize = blockDim.x;
    int gsize = gridDim.x;
    temp[tid] = 0;
    __syncthreads();

    // Calculate the global thread index and stride
    int i = tid + bid * bsize;
    int stride = bsize * gsize;

    // Perform atomic additions to the shared memory histogram
    while (i < size) {
        atomicAdd(&temp[buffer[i]], 1);
        i += stride;
    }
    __syncthreads();

    // Update the global histogram with values from the shared memory
    atomicAdd(&histo[tid], temp[tid]);
}