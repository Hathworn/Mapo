#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int addem(int a, int b) {
    return a + b;
}

__global__ void add(int a, int b, int *c) {
    // Utilize shared memory for faster access
    __shared__ int temp;
    
    if (threadIdx.x == 0) {
        temp = addem(a, b);
    }
    
    // Ensure all threads sync here before writing to global memory
    __syncthreads();
    
    if (threadIdx.x == 0) {
        *c = temp;
    }
}