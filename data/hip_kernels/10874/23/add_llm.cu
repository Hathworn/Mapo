#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add( int *a, int *b, int *c ) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // Compute global thread ID
    if (tid < N)
        c[tid] = a[tid] + b[tid];  // Perform addition if within bounds
}