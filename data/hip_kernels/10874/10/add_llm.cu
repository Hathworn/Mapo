#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c) {
    // Use shared memory to reduce global memory writes
    __shared__ int result;
    result = a + b;
    __syncthreads();
    if (threadIdx.x == 0) {
        *c = result;
    }
}