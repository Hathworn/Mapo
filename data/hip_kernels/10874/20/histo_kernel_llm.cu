#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo) {
    // Shared memory for atomic operations
    __shared__ unsigned int localHisto[256];
    
    // Initialize shared memory
    if (threadIdx.x < 256) {
        localHisto[threadIdx.x] = 0;
    }
    __syncthreads();
    
    // Calculate the starting index and the offset to the next block that each thread will be processing
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    while (i < size) {
        atomicAdd(&localHisto[buffer[i]], 1);
        i += stride;
    }
    
    __syncthreads();
    
    // Transfer results from shared memory to global memory
    if (threadIdx.x < 256) {
        atomicAdd(&histo[threadIdx.x], localHisto[threadIdx.x]);
    }
}