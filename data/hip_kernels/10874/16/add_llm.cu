#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // compute global thread ID
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}