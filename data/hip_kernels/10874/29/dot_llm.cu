#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(int size, float *a, float *b, float *c) {
    extern __shared__ float cache[]; // Use dynamic shared memory for better flexibility
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    
    float temp = 0.0f;
    // Use unrolling for improved performance
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        temp += a[i] * b[i];
    }

    // Set the cache values
    cache[cacheIndex] = temp;

    // Synchronize threads in this block
    __syncthreads();

    // Perform reduction in shared memory
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}