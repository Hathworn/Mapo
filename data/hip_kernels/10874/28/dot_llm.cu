#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(int size, float *a, float *b, float *c) {
    extern __shared__ float cache[]; // Use dynamic shared memory allocation
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0f; // Initialize to 0.0f for clarity
    while (tid < size) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    cache[cacheIndex] = temp;

    __syncthreads();

    // Use loop unrolling for reduction
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}