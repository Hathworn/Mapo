#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void pathAdjacencyKernel(int noTransitions, int noSegments, float* XY1, float* XY2, float* X4_X3, float* Y4_Y3, float* X2_X1, float* Y2_Y1, int* adjacency) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= noTransitions * noSegments) return;  // Early exit for out-of-bounds indices

    // Calculate segment indices
    int seg1 = idx / noSegments;
    int seg2 = idx % noSegments;

    // Calculate differences
    float Y1_Y3 = XY1[seg1 + noTransitions] - XY2[seg2 + noSegments];
    float X1_X3 = XY1[seg1] - XY2[seg2];

    // Calculate numerator and denominator
    float numa = X4_X3[seg2] * Y1_Y3 - Y4_Y3[seg2] * X1_X3;
    float numb = X2_X1[seg1] * Y1_Y3 - Y2_Y1[seg1] * X1_X3;
    float deno = Y4_Y3[seg2] * X2_X1[seg1] - X4_X3[seg2] * Y2_Y1[seg1];

    // Calculate u_a and u_b
    float u_a = numa / deno;
    float u_b = numb / deno;

    // Determine adjacency; store the result
    adjacency[idx] = (int)((u_a >= 0.0f) && (u_a <= 1.0f) && (u_b >= 0.0f) && (u_b <= 1.0f));
}