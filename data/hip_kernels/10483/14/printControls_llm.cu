#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void printControls(int noPaths, int path, int nYears, int* controls) {
    int ii = threadIdx.x + blockIdx.x * blockDim.x;  // Calculate global thread index
    if (ii < nYears) {  // Ensure within bounds
        printf("%d %d\n", ii, controls[path * nYears + ii]);
    }
}