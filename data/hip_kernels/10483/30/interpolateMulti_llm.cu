#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void interpolateMulti(int points, int noDims, int dimRes, float* surrogate, float* predictors, float* results) {
    // Global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < points) {
        // Utilize shared memory for frequently accessed data
        extern __shared__ float sharedMemory[];
        float* lower = sharedMemory;
        float* upper = &lower[noDims];
        float* coeffs = &upper[noDims];

        int lowerInd[MAX_DIMS];  // Assuming MAX_DIMS is defined appropriately

        for (int jj = 0; jj < noDims; jj++) {
            float lowerValue = surrogate[jj * dimRes];
            float upperValue = surrogate[(jj + 1) * dimRes - 1];
            lower[jj] = lowerValue;
            upper[jj] = upperValue;

            float predictorValue = predictors[noDims * idx + jj];
            lowerInd[jj] = (int)((dimRes - 1) * (predictorValue - lowerValue) / (upperValue - lowerValue));

            if (lowerInd[jj] >= (dimRes - 1)) {
                lowerInd[jj] = dimRes - 2;
            } else if (lowerInd[jj] < 0) {
                lowerInd[jj] = 0;
            }
        }

        // Uppermost dimensions x value
        float x0 = surrogate[lowerInd[0]];
        float x1 = surrogate[lowerInd[0] + 1];
        float xd = (predictors[noDims * idx] - x0) / (x1 - x0);

        int idxL = 0;

        // Optimize power calculations by avoiding redundant calls
        int dimFactor = (int)pow(2, noDims - 1);

        for (int jj = 0; jj < dimFactor; jj++) {
            idxL = dimRes * noDims;  // Reset for each iteration

            for (int kk = 1; kk < noDims; kk++) {
                int rem = ((jj >> (noDims - kk - 1)) + 1) & 1;
                idxL += (lowerInd[kk] + rem) * (int)pow(dimRes, noDims - kk - 1);
            }

            int idxU = idxL + (lowerInd[0] + 1) * (int)pow(dimRes, noDims - 1);
            idxL += lowerInd[0] * (int)pow(dimRes, noDims - 1);

            coeffs[jj] = surrogate[idxL] * (1 - xd) + surrogate[idxU] * xd;
        }

        // Compute interpolated value
        for (int jj = 1; jj < noDims; jj++) {
            x0 = surrogate[jj * dimRes + lowerInd[jj]];
            x1 = surrogate[jj * dimRes + lowerInd[jj] + 1];
            xd = (predictors[noDims * idx + jj] - x0) / (x1 - x0);

            int jump = (int)pow(2, noDims - jj - 2);
            for (int kk = 0; kk < (int)pow(2, jj); kk++) {
                coeffs[kk] = coeffs[kk] * (1 - xd) + coeffs[kk + jump] * xd;
            }
        }

        // Store the result
        results[idx] = coeffs[0];
    }
}
```
