#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void buildGlobalLinReg(int noPoints, int noDims, int dimRes, int nYears, int noControls, int year, int control, float* regCoeffs, float* xmins, float* xmaxes, float* regression) {

    // Global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < noPoints) {
        // Preallocate memory for indices and query point coordinates in registers
        int dimIdx[10]; // Assume noDims <= 10 for register allocation
        float xQ[10];   // Assume noDims <= 10 for register allocation

        int rem = idx;

        // Deconstruct the index into the index along each dimension
        for (int ii = 0; ii < noDims; ii++) {
            int div = (int)(rem / powf(dimRes, noDims - ii - 1));
            dimIdx[ii] = div;
            rem = rem - div * powf(dimRes, noDims - ii - 1);
        }

        // Get the query point coordinates
        for (int ii = 0; ii < noDims; ii++) {
            xQ[ii] = ((float)dimIdx[ii]) * (xmaxes[control * noDims + ii] -
                xmins[control * noDims + ii]) / (float)dimRes +
                xmins[control * noDims + ii];
        }

        // Use the regression coefficients to compute the value at this query point
        float computed = regCoeffs[0];

        for (int ii = 0; ii < noDims; ii++) {
            computed += xQ[ii] * regCoeffs[ii + 1];
        }

        if (computed >= 0) {
            computed = 0;
        }

        regression[year * noControls * (dimRes * noDims + (int)powf(dimRes, noDims) * 2)
            + control * (dimRes * noDims + (int)powf(dimRes, noDims) * 2) + dimRes *
            noDims + idx] = computed;
    }
}