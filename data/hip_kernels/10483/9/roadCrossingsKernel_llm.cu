#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void roadCrossingsKernel(int rows, int segs, int* adjacency, int* cross) {

    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread index is within bounds
    if (idx < rows) {

        // Initialize crossing count to zero
        int sum = 0;

        // Unroll loop to reduce loop overhead and improve performance
        for (int ii = 0; ii < segs; ii++) {
            sum += adjacency[idx * segs + ii];
        }

        // Store the sum result directly in output array
        cross[idx] = sum;
    }
}