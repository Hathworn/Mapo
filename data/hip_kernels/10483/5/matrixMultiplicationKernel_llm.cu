#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void matrixMultiplicationKernel(float *A, float* B, float* C, int a, int b, int d) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int ROW = by * blockDim.y + ty;
    int COL = bx * blockDim.x + tx;

    // Check if the thread is within matrix dimensions
    if (ROW < a && COL < d) {
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

        float Cv = 0;

        int aBegin = a * BLOCK_SIZE * by;
        int aStep = BLOCK_SIZE;
        int aEnd = aBegin + a - 1;

        int bBegin = BLOCK_SIZE * bx;
        int bStep = BLOCK_SIZE * d;

        for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
            // Load data from global to shared memory
            if (a + tx < a * (by + 1) && a / aStep < aEnd / aStep + 1) {
                As[ty][tx] = A[a + a * ty + tx];
            } else {
                As[ty][tx] = 0.0;
            }

            if (b + BLOCK_SIZE * ty + tx < b + d) {
                Bs[ty][tx] = B[b + BLOCK_SIZE * ty + tx];
            } else {
                Bs[ty][tx] = 0.0;
            }

            __syncthreads();

            // Accumulate results
            for (int k = 0; k < BLOCK_SIZE; ++k) {
                Cv += As[ty][k] * Bs[k][tx];
            }

            __syncthreads();
        }

        // Write results back to global memory
        if (ROW < a && COL < d) {
            C[ROW * d + COL] = Cv;
        }
    }
}