#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void mteKernel(int noPaths, int nYears, int noPatches, float timeStep, float* rgr, float* brownians, float* jumpSizes, float* jumps, float* speciesParams, float *initPops, float* caps, float*mmm, int* rowIdx, int* elemsPerCol, float* pathPops, float* eps) {
    // Calculate global thread index
    int ii = blockIdx.x*blockDim.x + threadIdx.x;

    // Check bounds
    if (ii < noPaths) {
        // Initialize initial population vector
        for (int jj = 0; jj < noPatches; jj++) {
            pathPops[(ii*2)*noPatches+jj] = initPops[jj];
        }

        float grMean = speciesParams[0];

        for (int jj = 0; jj < nYears; jj++) {
            // Use shared memory for partial results
            extern __shared__ float sharedPathPops[];
            
            // Movement and mortality initialization
            for (int kk = 0; kk < noPatches; kk++) {
                sharedPathPops[kk] = 0.0;
            }

            __syncthreads(); // Ensure shared memory is initialized

            int iterator = 0;
            for (int kk = 0; kk < noPatches; kk++) {
                for (int ll = 0; ll < elemsPerCol[kk]; ll++) {
                    sharedPathPops[kk] += pathPops[(ii*2)*noPatches+rowIdx[iterator]] * mmm[iterator];
                    iterator++;
                }
            }

            __syncthreads(); // Wait for all threads to complete

            // Update pathPops from shared memory
            for (int kk = 0; kk < noPatches; kk++) {
                pathPops[(ii*2+1)*noPatches+kk] = sharedPathPops[kk];
            }

            // Natural birth and death calculations
            float jump = (jumps[ii*nYears + jj] < speciesParams[6]) ? 1.0f : 0.0f;
            float meanP = speciesParams[1];
            float reversion = speciesParams[4];

            float brownian = brownians[ii*nYears + jj] * speciesParams[2];
            float jumpSize = jumpSizes[ii*nYears + jj] * powf(speciesParams[5], 2) - powf(speciesParams[5], 2) / 2;

            grMean = grMean + reversion * (meanP - grMean) * timeStep + grMean * brownian + (expf(jumpSize) - 1) * grMean * jump;

            for (int kk = 0; kk < noPatches; kk++) {
                float gr = speciesParams[7] * rgr[ii*(nYears*noPatches) + jj*noPatches + kk] * grMean + grMean;
                pathPops[(ii*2)*noPatches+kk] = pathPops[(ii*2+1)*noPatches+kk] * (1.0f + gr * (caps[kk] - pathPops[(ii*2+1)*noPatches+kk]) / caps[kk]);
            }
        }

        // Calculate epsilon values
        eps[ii] = 0.0f;
        for (int jj = 0; jj < noPatches; jj++) {
            eps[ii] += pathPops[(ii*2+1)*noPatches+jj];
        }
    }
}