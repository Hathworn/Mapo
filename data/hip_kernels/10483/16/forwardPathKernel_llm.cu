#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void forwardPathKernel(int noPaths, int nYears, int noSpecies, int noPatches, int noControls, int noUncertainties, float timeStep, float* initPops, float* pops, float* mmm, int* rowIdx, int* elemsPerCol, int maxElems, float* speciesParams, float* caps, float* aars, float* uncertParams, int* controls, float* uJumps, float* uBrownian, float* uJumpSizes, float* uJumpsSpecies, float* uBrownianSpecies, float* uJumpSizesSpecies, float* rgr, float* uResults, float* totalPops) {

    // Global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check thread boundary
    if (idx >= noPaths) return;

    // Use shared memory for species parameters
    extern __shared__ float sharedMem[];
    float* grMean = sharedMem;

    // Initialise the population data at time t=0
    for (int ii = 0; ii < noSpecies; ii++) {
        float population = 0;
        for (int jj = 0; jj < noPatches; jj++) {
            pops[idx*(nYears+1)*noSpecies*noPatches + ii*noPatches + jj] = initPops[jj];
            population += pops[idx*(nYears+1)*noSpecies*noPatches + ii*noPatches + jj];
        }
        totalPops[idx*(nYears+1)*noSpecies + ii] = population;
    }

    // Carry over the initial value for all uncertainties
    for (int ii = 0; ii < noUncertainties; ii++) {
        uResults[idx*noUncertainties*(nYears+1) + ii] = uncertParams[ii*6];
    }

    // Initialize grMean using shared memory
    for (int ii = 0; ii < noSpecies; ii++) {
        grMean[ii] = speciesParams[ii*8];
    }

    // All future time periods
    for (int ii = 0; ii < nYears; ii++) {
        // Control to pick
        int control = controls[idx*nYears + ii];

        for (int jj = 0; jj < noSpecies; jj++) {
            totalPops[idx*(nYears+1)*noSpecies + (ii+1)*noSpecies + jj] = 0;

            // Adjust the global growth rate mean
            float jump = (uJumpsSpecies[idx*noSpecies*nYears + ii*noSpecies + jj] < speciesParams[jj*8 + 5]) ? 1.0f : 0.0f;
            float meanP = speciesParams[jj*8 + 1];
            float reversion = speciesParams[jj*8 + 4];
            float brownian = uBrownianSpecies[idx*noSpecies*nYears + ii*noSpecies + jj] * speciesParams[jj*8 + 2];
            float jumpSize = uJumpSizesSpecies[idx*noSpecies*nYears + ii*noSpecies + jj] * powf(speciesParams[jj*8 + 5], 2) - powf(speciesParams[jj*8 + 5], 2) / 2;

            grMean[jj] = grMean[jj] + reversion * (meanP - grMean[jj]) * timeStep + grMean[jj] * brownian + (expf(jumpSize) - 1) * grMean[jj] * jump;

            float initialPopulation = 0.0f;
            for (int kk = 0; kk < noPatches; kk++) {
                initialPopulation += pops[idx*(nYears+1)*noSpecies*noPatches + ii*noSpecies*noPatches + jj*noPatches + kk];
            }

            for (int kk = 0; kk < noControls; kk++) {
                float totalPop = 0.0f;
                int iterator = 0;

                for (int ll = 0; ll < noPatches; ll++) {
                    float population = 0.0f;
                    for (int mm = 0; mm < elemsPerCol[(jj*noControls + kk)*noPatches + ll]; mm++) {
                        float value = pops[idx*(nYears+1)*noSpecies*noPatches + ii*noSpecies*noPatches + jj*noPatches + rowIdx[iterator + (jj*noControls + kk)*maxElems]] * mmm[iterator + (jj*noControls + kk)*maxElems];
                        population += value;
                        iterator++;
                    }
                    totalPop += population;

                    if (kk == control && ii < nYears) {
                        rgr[idx*noSpecies*noPatches*nYears + ii*noSpecies*noPatches + jj*noPatches + ll] = grMean[jj] + rgr[idx*noSpecies*noPatches*nYears + ii*noSpecies*noPatches + jj*noPatches + ll] * speciesParams[jj*8 + 7];
                        float gr = rgr[idx*noSpecies*noPatches*nYears + ii*noSpecies*noPatches + jj*noPatches + ll];

                        pops[idx*(nYears+1)*noSpecies*noPatches + (ii+1)*noSpecies*noPatches + jj*noPatches + ll] = population * (1.0f + gr*(caps[jj*noPatches + ll] - population) / caps[jj*noPatches + ll] / 100.0);
                        totalPops[idx*noSpecies*(nYears+1) + (ii+1)*noSpecies + jj] += pops[idx*(nYears+1)*noSpecies*noPatches + (ii+1)*noSpecies*noPatches + jj*noPatches + ll];
                    }
                }
                aars[idx*(nYears+1)*noControls*noSpecies + ii*noControls*noSpecies + jj*noControls + kk] = totalPop / initialPopulation;
            }
        }

        for (int jj = 0; jj < noUncertainties; jj++) {
            float jump = (uJumps[idx*noUncertainties*nYears + ii*noUncertainties + jj] < uncertParams[jj*6 + 5]) ? 1.0f : 0.0f;
            float curr = uResults[idx*noUncertainties*(nYears+1) + ii*noUncertainties + jj];
            float meanP = uncertParams[jj*6 + 1];
            float reversion = uncertParams[jj*6 + 3];
            float brownian = uBrownian[idx*noUncertainties*nYears + ii*noUncertainties + jj] * uncertParams[jj*6 + 2];
            float jumpSize = uJumpSizes[idx*noUncertainties*nYears + ii*noUncertainties + jj] * powf(uncertParams[jj*6 + 4], 2) - powf(uncertParams[jj*6 + 4], 2) / 2;

            uResults[idx*noUncertainties*(nYears+1) + (ii+1)*noUncertainties + jj] = curr + reversion * (meanP - curr) * timeStep + curr * brownian + (expf(jumpSize) - 1) * curr * jump;
        }
    }
}