#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void matrixDivisionKernelEW(const float* A, const float* B, float* C, int a, int b) {
    // Calculate the unique index for each thread to avoid accessing out of bounds
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index calculation correctly reflects the grid's dimensions
    int index = ROW * b + COL; // Using 'b' for column in calculation

    if (ROW < a && COL < b) {
        // Simplify index arithmetic and ensure indices are valid
        C[index] = A[index] / B[index];
    }
}