#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

// Optimized kernel using shared memory
__global__ void computeStateMinMax(int noControls, int noDims, int noPaths, int* dataPoints, float* xvals, float* xmins, float* xmaxes) {
    extern __shared__ float sharedMem[];
    float* xmin = sharedMem;
    float* xmax = sharedMem + noDims;

    int tid = threadIdx.x;
    for (int ii = blockIdx.x; ii < noControls; ii += gridDim.x) {
        if (tid < noDims) {
            xmin[tid] = FLT_MAX;
            xmax[tid] = -FLT_MAX;
        }
        __syncthreads();

        if (tid < dataPoints[ii]) {
            for (int jj = 0; jj < noDims; jj++) {
                float xtemp = xvals[ii * noDims * noPaths + jj * noPaths + tid];
                atomicMin(&xmin[jj], xtemp);
                atomicMax(&xmax[jj], xtemp);
            }
        }
        __syncthreads();

        if (tid < noDims) {
            xmins[ii * noDims + tid] = xmin[tid];
            xmaxes[ii * noDims + tid] = xmax[tid];
        }
        __syncthreads();
    }

    if (blockIdx.x == 0 && tid < noDims) {
        xmins[noControls * noDims + tid] = xmin[tid];
        xmaxes[noControls * noDims + tid] = xmax[tid];

        for (int ii = 1; ii < noControls; ii++) {
            float xtemp = xmins[ii * noDims + tid];
            atomicMin(&xmins[noControls * noDims + tid], xtemp);

            xtemp = xmaxes[ii * noDims + tid];
            atomicMax(&xmaxes[noControls * noDims + tid], xtemp);
        }
    }
}