#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void createQueryPoints(int noPoints, int noDims, int dimRes, int control, int noControls, int year, float* xmins, float* xmaxes, float* regression, float* queryPts) {
    // Global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < noPoints) {
        // Allocate on stack instead of heap for performance gain
        int dimIdx[MAX_DIMS];  // Assuming MAX_DIMS is defined and encompasses all possible dimensions

        int rem = idx;

        // Precompute powers of dimRes to avoid repeated calculations
        int powers[MAX_DIMS];
        for (int ii = 0; ii < noDims; ii++) {
            powers[ii] = pow(dimRes, noDims - ii - 1);
        }

        for (int ii = 0; ii < noDims; ii++) {
            int div = rem / powers[ii];
            dimIdx[ii] = div;
            rem -= div * powers[ii];
        }

        // Get the query point coordinates using precomputed values where possible
        for (int ii = 0; ii < noDims; ii++) {
            float range = xmaxes[noControls * noDims + ii] - xmins[noControls * noDims + ii];
            queryPts[idx + ii * noPoints] = ((float)dimIdx[ii]) * range / (float)(dimRes - 1) + xmins[noControls * noDims + ii];

            // Save the X value for the query point
            regression[year * noControls * (dimRes * noDims + (int)pow(dimRes, noDims) * 2) + control * (dimRes * noDims + (int)pow(dimRes, noDims) * 2) + ii * dimRes + dimIdx[ii]] = queryPts[idx + ii * noPoints];
        }
    }
}