#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void matrixMultiplicationKernelEW(const float* A, const float* B, float* C, int a, int b) {
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if ROW and COL are within bounds
    if (ROW < a && COL < b) {
        // Optimize by calculating index once
        int index = ROW * b + COL;
        C[index] = A[index] * B[index];
    }
}