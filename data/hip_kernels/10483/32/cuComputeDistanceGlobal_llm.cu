#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuComputeDistanceGlobal( float* A, int wA, int pA, float* B, int wB, int pB, int dim,  float* AB){

// Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
__shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
__shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

// Thread index
int tx = threadIdx.x;
int ty = threadIdx.y;

// Block index
int row = BLOCK_DIM * blockIdx.y + ty;
int col = BLOCK_DIM * blockIdx.x + tx;

// Other variables
float tmp;
float ssd = 0;

// Loop over all the sub-matrices of A and B required to compute the block sub-matrix
for (int m = 0; m < dim; m += BLOCK_DIM) {
    
  // Load the matrices from device memory to shared memory; each thread loads one element of each matrix
  if ((row < wA) && (m + tx < dim))
      shared_A[ty][tx] = A[row * pA + m + tx];
  else
      shared_A[ty][tx] = 0;
  
  if ((col < wB) && (m + ty < dim))
      shared_B[ty][tx] = B[(m + ty) * pB + col];
  else
      shared_B[ty][tx] = 0;

  // Synchronize to make sure the matrices are loaded
  __syncthreads();

  // Compute the difference between the two matrices; each thread computes one element of the block sub-matrix
  for (int k = 0; k < BLOCK_DIM; ++k) {
    tmp = shared_A[ty][k] - shared_B[k][tx];
    ssd += tmp*tmp;
  }

  // Synchronize to make sure that the preceding computation is done before loading two new sub-matrices 
  __syncthreads();
}

// Write the block sub-matrix to device memory; each thread writes one element
if (row < wA && col < wB)
  AB[row * pB + col] = ssd;
}