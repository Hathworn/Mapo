#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void matrixMultiplicationKernelNaive(const float* A, const float* B, float* C, int a, int b, int c, int d) {

    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    float tmpSum = 0.0f;

    if (ROW < a && COL < d) {
        // Use shared memory to optimize for coalesced memory access
        extern __shared__ float shared[];
        float* As = shared;
        float* Bs = (float*)&shared[b * blockDim.y];

        for (int t = 0; t < (b - 1) / blockDim.x + 1; ++t) {
            if (t * blockDim.x + threadIdx.x < b && ROW < a)
                As[threadIdx.y * blockDim.x + threadIdx.x] = A[ROW * b + t * blockDim.x + threadIdx.x];
            else
                As[threadIdx.y * blockDim.x + threadIdx.x] = 0.0f;

            if (t * blockDim.y + threadIdx.y < c && COL < d)
                Bs[threadIdx.y * blockDim.x + threadIdx.x] = B[(t * blockDim.y + threadIdx.y) * d + COL];
            else
                Bs[threadIdx.y * blockDim.x + threadIdx.x] = 0.0f;

            // Synchronize to make sure the sub-matrices are loaded
            __syncthreads();

            for (int ii = 0; ii < blockDim.x; ++ii) {
                tmpSum += As[threadIdx.y * blockDim.x + ii] * Bs[ii * blockDim.x + threadIdx.x];
            }

            // Synchronize to prevent data hazard
            __syncthreads();
        }
        C[ROW * d + COL] = tmpSum;
    }
}