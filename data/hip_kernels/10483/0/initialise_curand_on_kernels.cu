#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void initialise_curand_on_kernels(hiprandState* state, unsigned long seed) {
int idx = blockIdx.x*blockDim.x+threadIdx.x;
hiprand_init(seed, idx, 0, &state[idx]);
}