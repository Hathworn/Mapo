#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void initialise_curand_on_kernels(hiprandState* state, unsigned long seed) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < gridDim.x * blockDim.x) { // Ensure idx is within the bounds
        hiprand_init(seed, idx, 0, &state[idx]);
    }
}