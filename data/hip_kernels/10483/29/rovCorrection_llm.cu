#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void rovCorrection(int noPoints, int noDims, int dimRes, int nYears, int noControls, int year, int control, float* regression) {
    // Global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < noPoints) {
        // Precalculate constant indices for performance
        int baseIndex = year * noControls * (dimRes * noDims + (int)powf(dimRes, noDims) * 2) + control * (dimRes * noDims + (int)powf(dimRes, noDims) * 2);
        int regIndex = baseIndex + dimRes * noDims + idx;

        float currVal = regression[regIndex];

        // Ensure surrogate value is non-positive
        if (currVal > 0) {
            regression[regIndex] = 0.0;
        }
    }
}