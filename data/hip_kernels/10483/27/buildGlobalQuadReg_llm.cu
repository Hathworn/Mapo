#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void buildGlobalQuadReg(int noPoints, int noDims, int dimRes, int nYears, int noControls, int year, int control, float* regCoeffs, float* xmins, float* xmaxes, float* regression) {

    // Global thread index
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx < noPoints) {
        // Allocate memory on stack for dimIdx and xQ
        int dimIdx[MAX_DIMS]; // Assuming a maximum number of dimensions
        float xQ[MAX_DIMS]; // Assuming a maximum number of dimensions

        // Deconstruct the index into the index along each dimension
        int rem = idx;
        for (int ii = 0; ii < noDims; ii++) {
            int div = rem / (int)pow(dimRes, noDims - ii - 1);
            dimIdx[ii] = div;
            rem -= div * (int)pow(dimRes, noDims - ii - 1);
        }

        // Get the query point coordinates
        for (int ii = 0; ii < noDims; ii++) {
            xQ[ii] = ((float)dimIdx[ii]) * (xmaxes[control * noDims + ii] -
                      xmins[control * noDims + ii]) / (float)dimRes +
                     xmins[control * noDims + ii];
        }

        // Use the regression coefficients to compute the value at this query point
        // Constant
        float computed = regCoeffs[0];

        // Linear Terms
        for (int ii = 0; ii < noDims; ii++) {
            computed += xQ[ii] * regCoeffs[ii + 1];
        }

        // Quadratic and Interacting Terms
        int counter = 0;
        for (int ii = 0; ii < noDims; ii++) {
            for (int jj = ii; jj < noDims; jj++) {
                computed += xQ[ii] * xQ[jj] * regCoeffs[counter + 1 + noDims];
                counter++;
            }
        }

        // Payoff adjustment
        computed = (computed >= 0.0f) ? 0.0f : computed;

        // Store result in regression array
        int baseIdx = year * noControls * (dimRes * noDims + (int)pow(dimRes, noDims) * 2)
                      + control * (dimRes * noDims + (int)pow(dimRes, noDims) * 2)
                      + dimRes * noDims + idx;
        regression[baseIdx] = computed;
    }
}