#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void computePathStates(int noPaths, int noDims, int nYears, int noControls, int year, float unitCost, float unitRevenue, int* controls, int noFuels, float *fuelCosts, float *uResults, float *uComposition, int noUncertainties, int *fuelIdx, int noCommodities, float* aars, float* totalPops, float* xin, int* currControls) {
    
    // Global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < noPaths) {
        
        // 1. Adjusted population for each species
        for (int ii = 0; ii < noDims - 1; ii++) {
            // Use shared memory to minimize repeated global memory load
            extern __shared__ float sharedPops[];
            sharedPops[threadIdx.x * (noDims - 1) + ii] = totalPops[idx * (noDims - 1) * (nYears + 1) + year * (noDims - 1) + ii];
            __syncthreads();  // Synchronize threads in block
            xin[idx * noDims + ii] = sharedPops[threadIdx.x * (noDims - 1) + ii] * aars[idx * (nYears + 1) * noControls * (noDims - 1) + year * noControls * (noDims - 1) + ii * noControls + (noControls - 1)];
        }
        
        // 2. Unit profit
        float unitFuel = 0.0;
        float orePrice = 0.0;
        
        // Compute the unit fuel cost component
        for (int ii = 0; ii < noFuels; ii++) {
            unitFuel += fuelCosts[ii] * uResults[idx * (nYears + 1) * noUncertainties + (year) * noUncertainties + fuelIdx[ii]];
        }
        
        // Compute the unit revenue from ore
        for (int ii = 0; ii < noCommodities; ii++) {
            orePrice += uComposition[idx * nYears * noCommodities + (year) * noCommodities + ii] * uResults[idx * (nYears + 1) * noUncertainties + (year) * noUncertainties + noFuels + ii];
        }
        
        xin[idx * noDims + noDims - 1] = unitCost + unitFuel - unitRevenue * orePrice;
        currControls[idx] = controls[idx * nYears + year];
    }
}