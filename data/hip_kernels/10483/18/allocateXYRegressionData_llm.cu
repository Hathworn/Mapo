#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void allocateXYRegressionData(int noPaths, int noControls, int noDims, int nYears, float* speciesParams, int year, int* controls, float* xin, float *condExp, int *dataPoints, float *xvals, float *yvals) {

    // Use shared memory for storing dataPoints to reduce global memory access
    extern __shared__ int sharedDataPoints[];

    int tid = threadIdx.x;
    if (tid < noControls) {
        sharedDataPoints[tid] = 0;
    }
    __syncthreads();

    int ii = blockIdx.x * blockDim.x + tid;
    if (ii < noPaths) {
        if (controls[ii] >= noControls) {
            printf("Invalid control %d\n", controls[ii]);
        }

        bool valid = true;
        for (int jj = 0; jj < (noDims - 1); jj++) {
            if (xin[ii * noDims + jj] < speciesParams[8 * jj + 3]) {
                valid = false;
                break;
            }
        }

        if (valid || controls[ii] == 0) {
            int controlIdx = controls[ii];
            int dataPointIdx = atomicAdd(&sharedDataPoints[controlIdx], 1);

            // Save the conditional expectation
            yvals[noPaths * controlIdx + dataPointIdx] = condExp[(year + 1) * noPaths + ii];

            // Save the input dimension values to the corresponding data group
            for (int jj = 0; jj < noDims; jj++) {
                xvals[controlIdx * noPaths * noDims + jj * noPaths + dataPointIdx] = xin[ii * noDims + jj];
            }
        }
    }

    __syncthreads();
    // Copy data from shared to global memory after processing
    if (tid < noControls) {
        atomicAdd(&dataPoints[tid], sharedDataPoints[tid]);
    }
}