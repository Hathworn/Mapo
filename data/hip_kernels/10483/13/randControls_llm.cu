#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void randControls(int noPaths, int nYears, int noControls, float* randCont, int* control) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = gridDim.x * blockDim.x;
    
    // Loop to handle more elements per thread for larger datasets
    for (int i = idx; i < noPaths * nYears; i += totalThreads) {
        int ctrlValue = (int)(randCont[i] * noControls);
        control[i] = (ctrlValue == noControls) ? (noControls - 1) : ctrlValue;
    }
}