#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void matrixElementWiseMultiplicationKernelOptimized(const float* A, const float* B, float* C, int a, int b) {

    // Efficient index calculation for 1D grid of blocks processing
    int idx = blockIdx.y * blockDim.y * b + threadIdx.y * b + blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid redundant calculations, improve memory coalescing
    if (idx < a * b) {
        C[idx] = A[idx] * B[idx];
    }
}