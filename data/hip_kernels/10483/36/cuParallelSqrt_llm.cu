#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuParallelSqrt(float *dist, int width, int pitch, int k) {
    // Calculate global index
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Calculate linear index and check bounds
    if (xIndex < width && yIndex < k) {
        unsigned int index = yIndex * pitch + xIndex;

        // Use in-place update
        dist[index] = sqrtf(dist[index]);
    }
}