#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void patchComputation(int noCandidates, int W, int H, int skpx, int skpy, int xres, int yres, float subPatchArea, float xspacing, float yspacing, float capacity, int uniqueRegions, const int* labelledImage, const float* pops, float* results) {

    // Get global index of thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < noCandidates) {
        // Dimensions arranged as X->Y->R
        int rem = idx;
        int blockIdxY = idx / (xres * uniqueRegions);
        rem -= blockIdxY * (xres * uniqueRegions);
        int blockIdxX = rem / uniqueRegions;
        rem -= blockIdxX * uniqueRegions;
        // Valid region numbering starts at 1, not 0
        int regionNo = rem + 1;

        int blockSizeX = min(skpx, H - blockIdxX * skpx);  // Simplified block size calculation
        int blockSizeY = min(skpy, W - blockIdxY * skpy);

        // Precompute repeated multiplications outside the loop
        int startX = blockIdxX * skpx;
        int startY = blockIdxY * skpy;

        // Iterate through each sub patch for this large grid cell
        float area = 0.0f, cap = 0.0f, pop = 0.0f;
        float cx = 0.0f, cy = 0.0f;

        for (int ii = 0; ii < blockSizeX; ii++) {
            for (int jj = 0; jj < blockSizeY; jj++) {
                int xCoord = startX + ii;
                int yCoord = startY + jj;

                if (labelledImage[xCoord + yCoord * W] == regionNo) {
                    area += 1.0f;  // Convert boolean directly to float
                    pop += pops[xCoord + yCoord * W];
                    cx += ii;
                    cy += jj;
                }
            }
        }

        if (area > 0) {
            cx = xspacing * (cx / area + startX);
            cy = yspacing * (cy / area + startY);
            area *= subPatchArea;
            cap = area * capacity;
        }

        // Store results to output matrix
        results[5 * idx] = area;
        results[5 * idx + 1] = cap;
        results[5 * idx + 2] = pop;
        results[5 * idx + 3] = cx;
        results[5 * idx + 4] = cy;
    }
}