#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void expPVPath(const int noPaths, const float gr, const int nYears, const float meanP, const float timeStep, const float rrr, float current, float reversion, float jumpProb, const float* brownian, const float* jumpSize, const float* jump, float* result) {

    // Get the global index for the matrix
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < noPaths) {
        // Use cache to store repeated computation
        float pow_gr_rrr[nYears];  // Precompute pow values for optimization
        for (int ii = 0; ii < nYears; ii++) {
            pow_gr_rrr[ii] = pow(1 + gr, ii) / pow((1 + rrr), ii);
        }

        // Simulate a forward path
        float value = 0;
        float curr = current;

        for (int ii = 0; ii < nYears; ii++) {
            // Use ternary operation to simplify jumped evaluation
            float jumped = (jump[idx + ii] < jumpProb) ? 1.0f : 0.0f;

            // Simplified calculation into single expression
            curr += reversion * (meanP - curr) * timeStep + curr * brownian[idx + ii] +
                    (exp(jumpSize[idx + ii]) - 1) * curr * jumped;
            value += pow_gr_rrr[ii] * curr;
        }

        result[idx] = value;
    }
}