#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void firstPeriodInduction(int noPaths, int nYears, int noSpecies, int noControls, float timeStep, float unitCost, float unitRevenue, float rrr, int noFuels, int noCommodities, float* Q, float* fuelCosts, float* totalPops, float* speciesParams, int* controls, float* aars, float* uComposition, float* uResults, int* fuelIdx, float* condExp, int* optCont, float* stats) {

    extern __shared__ float sharedMem[]; // Use shared memory for temporary storage
    float* payoffs = sharedMem;
    float* dataPoints = payoffs + noControls;
    bool* valid = (bool*)(dataPoints + noControls);

    int tid = threadIdx.x;
    if (tid < noControls) {
        payoffs[tid] = 0.0f;
        dataPoints[tid] = 0.0f;
        valid[tid] = true;
    }
    __syncthreads();

    float unitFuel = 0.0f, orePrice = 0.0f;
    if (tid < noFuels) {
        atomicAdd(&unitFuel, fuelCosts[tid] * uResults[fuelIdx[tid]]);
    }
    if (tid < noCommodities) {
        atomicAdd(&orePrice, uComposition[tid] * uResults[noFuels + tid]);
    }
    __syncthreads(); // Synchronize after using shared variables

    for (int ii = tid; ii < noPaths; ii += blockDim.x) {
        int control = controls[ii * nYears];
        atomicAdd(&payoffs[control], condExp[ii + noPaths]);
        atomicAdd(&dataPoints[control], 1.0f);
    }
    __syncthreads(); // Ensure all threads have added their payoffs

    float adjPop;
    for (int ii = tid; ii < noControls; ii += blockDim.x) {
        if (dataPoints[ii] > 0) {
            payoffs[ii] /= (dataPoints[ii] * (1 + rrr * timeStep / 100));
        } else {
            valid[ii] = false;
            continue;
        }

        for (int jj = 0; jj < noSpecies; jj++) {
            adjPop = totalPops[jj] * aars[jj * noControls + ii];
            if (adjPop < speciesParams[noSpecies * jj + 3] && ii > 0) {
                valid[ii] = false;
                break;
            }
        }
        if (valid[ii]) {
            payoffs[ii] += Q[ii] * (unitCost + unitFuel - unitRevenue * orePrice);
            if (payoffs[ii] > 0) payoffs[ii] = 0.0f;
        } else {
            payoffs[ii] = NAN;
        }
    }
    __syncthreads(); // Ensure all calculations are finished before finding the best control

    if (tid == 0) {
        float bestExp = payoffs[0];
        int bestCont = 0;
        for (int ii = 1; ii < noControls; ii++) {
            if (isfinite(payoffs[ii]) && payoffs[ii] < bestExp) {
                bestExp = payoffs[ii];
                bestCont = ii;
            }
        }

        stats[2] = 0;
        for (int ii = 0; ii < noPaths; ii++) {
            condExp[ii] = bestExp;
            optCont[ii] = bestCont;
            if (controls[ii * nYears] == bestCont) {
                stats[2] += (condExp[ii + noPaths] - payoffs[bestCont]) * (condExp[ii + noPaths] - payoffs[bestCont]);
            }
        }

        stats[0] = condExp[0];
        stats[1] = (float)optCont[0];
        stats[2] = sqrt(stats[2] / (dataPoints[bestCont] * (1 + rrr / (100 * timeStep))));
    }
}