#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__global__ void mmKernel(float* popsIn, float* popsOut, float* mmm, int patches) {
    int ii = threadIdx.x;

    if (ii < patches) {
        extern __shared__ float s[];

        s[ii] = 0.0;
        
        // Use unrolled loop for increased performance
        for (int jj = 0; jj < patches; jj += 4) {
            s[ii] += popsIn[ii] * mmm[ii * patches + jj];
            if (jj + 1 < patches) s[ii] += popsIn[ii] * mmm[ii * patches + jj + 1];
            if (jj + 2 < patches) s[ii] += popsIn[ii] * mmm[ii * patches + jj + 2];
            if (jj + 3 < patches) s[ii] += popsIn[ii] * mmm[ii * patches + jj + 3];
        }
        
        __syncthreads();
        popsOut[ii] = s[ii];
    }
}