#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR CHECKING MACROS //////////////////////////////////////////////////////

__device__ void solveLinearSystem(int dims, float *A, float *B, float *C) {
    // First generate upper triangular matrix for the augmented matrix
    float *swapRow;
    swapRow = (float*)malloc((dims+1)*sizeof(float));

    for (int ii = 0; ii < dims; ii++) {
        C[ii] = B[ii];
    }

    for (int ii = 0; ii < dims; ii++) {
        // Search for maximum in this column
        float maxElem = fabsf(A[ii*dims+ii]);
        int maxRow = ii;
    
        for (int jj = (ii+1); jj < dims; jj++) {
            if (fabsf(A[ii*dims+jj] > maxElem)) {
                maxElem = fabsf(A[ii*dims+jj]);
                maxRow = jj;
            }
        }
        
        // Swap maximum row with current row if needed
        if (maxRow != ii) {
            for (int jj = ii; jj < dims; jj++) {
                swapRow[jj] = A[jj*dims+ii];
                A[jj*dims+ii] = A[jj*dims+maxRow];
                A[jj*dims+maxRow] = swapRow[jj];
            }
            
            swapRow[dims] = C[ii];
            C[ii] = C[maxRow];
            C[maxRow] = swapRow[dims];
        }
        
        // Make all rows below this one 0 in current column
        for (int jj = (ii+1); jj < dims; jj++) {
            float factor = -A[ii*dims+jj]/A[ii*dims+ii];
            
            // Work across columns
            for (int kk = ii; kk < dims; kk++) {
                A[kk*dims+jj] += factor*A[kk*dims+ii];
            }
            
            // Results vector
            C[jj] += factor*C[ii];
        }
    }
    free(swapRow);
    
    // Solve equation for an upper triangular matrix
    for (int ii = dims-1; ii >= 0; ii--) {
        C[ii] = C[ii]/A[ii*dims+ii];
        
        for (int jj = ii-1; jj >= 0; jj--) {
            C[jj] -= C[ii]*A[ii*dims+jj];
        }
    }
}

// Optimized kernel function
__global__ void multiLocLinReg(int noPoints, int noDims, int dimRes, int nYears, int noControls, int year, int control, int k, int* dataPoints, float *xvals, float *yvals, float *regression, float* xmins, float* xmaxes, float *dist, int *ind) {
    
    // Global thread index
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    
    if (idx < noPoints) {
        if (dataPoints[control] < 3) {
            // Write zero if not enough data points for control
            regression[year*noControls*(dimRes*noDims + (int)pow(dimRes,noDims)*2) + control*(dimRes*noDims + (int)pow(dimRes,noDims)*2) + dimRes*noDims + idx] = 0.0;
        } else {
            // Use thread-local arrays instead of malloc
            int dimIdx[10]; // Assuming noDims <= 10
            float xQ[10];
            float A[121]; // Assuming (noDims + 1) <= 11, hence max (noDims+1)^2 is 121
            float B[11];  // Assuming (noDims + 1) <= 11
            float X[11];  // Assuming (noDims + 1) <= 11
        
            int rem = idx;
            for (int ii = 0; ii < noDims; ii++) {
                int div = rem / pow(dimRes, noDims-ii-1);
                dimIdx[ii] = div;
                rem -= div * pow(dimRes, noDims-ii-1);
            }
            
            for (int ii = 0; ii < noDims; ii++) {
                xQ[ii] = (float)dimIdx[ii]*(xmaxes[ii] - xmins[ii])/(float)(dimRes - 1) + xmins[ii];
            }
            
            // Bandwidth for kernel
            float h = dist[noPoints*(k-1) + idx];
        
            for (int ii = 0; ii <= noDims; ii++) {
                B[ii] = 0.0;
                for (int kk = 0; kk < k; kk++) {
                    float d = dist[noPoints*kk + idx];
                    // Epanechnikov kernel
                    float z = 0.75*(1-pow(d/h,2));

                    if (ii == 0) {
                        B[ii] += yvals[ind[noPoints*kk + idx] - 1]*z;
                    } else {
                        B[ii] += yvals[ind[noPoints*kk + idx] - 1]*(xvals[(ind[noPoints*kk + idx] - 1)*noDims + ii - 1] - xQ[ii-1])*z;
                    }
                }
                
                for (int jj = 0; jj <= noDims; jj++) {
                    A[jj*(noDims+1)+ii] = 0.0;
                    for (int kk = 0; kk < k; kk++) {
                        float d = dist[noPoints*kk + idx];
                        float z = 0.75*(1-pow(d/h,2));

                        if ((ii == 0) && (jj == 0)) {
                            A[jj*(noDims+1)+ii] += 1.0*z;
                        } else if (ii == 0) {
                            A[jj*(noDims+1)+ii] += (xvals[(ind[noPoints*kk + idx] - 1)*noDims + jj - 1] - xQ[jj - 1])*z;
                        } else if (jj == 0) {
                            A[jj*(noDims+1)+ii] += (xvals[(ind[noPoints*kk + idx] - 1)*noDims + ii - 1] - xQ[ii - 1])*z;
                        } else {
                            A[jj*(noDims+1)+ii] += (xvals[(ind[noPoints*kk + idx] - 1)*noDims + jj - 1] - xQ[jj-1])*(xvals[(ind[noPoints*kk + idx] - 1)*noDims + ii - 1] - xQ[ii - 1])*z;
                        }
                    }
                }
            }

            // Solve the linear system using LU decomposition.
            solveLinearSystem(noDims+1, A, B, X);

            // Save the regression result
            regression[year*noControls*(dimRes*noDims + (int)pow(dimRes,noDims)*2) + control*(dimRes*noDims + (int)pow(dimRes,noDims)*2) + dimRes*noDims + idx] = X[0];
        }
    }
}