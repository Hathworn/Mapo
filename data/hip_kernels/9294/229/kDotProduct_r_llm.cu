#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target,  const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f; // Use register for thread-local accumulation

    // Unrolling loop for efficiency
    if (eidx < numElements) {
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            sum += a[eidx] * b[eidx];
        }
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Optimize reduction by using a single warp for the final accumulation
    for (int offset = DP_BLOCKSIZE / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            shmem[threadIdx.x] += shmem[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Write the computed block result to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}