#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Compute global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Total number of threads
    const int numThreads = blockDim.x * gridDim.x;
    // Loop through all elements assigned to this thread with stride of total threads
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Compute target coordinates
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        // Map target coordinates to source coordinates using modulus
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Read from source and write to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}