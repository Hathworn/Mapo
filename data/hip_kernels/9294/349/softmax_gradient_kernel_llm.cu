#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    extern __shared__ float reduction_buffer[];
    const int idx = threadIdx.x;
    const int block_offset = blockIdx.x * dim;

    float partial_sum = 0.0f;

    // Compute partial dot product
    for (int i = idx; i < dim; i += blockDim.x) {
        partial_sum += dY[block_offset + i] * Y[block_offset + i];
    }
    reduction_buffer[idx] = partial_sum;
    __syncthreads();

    // Complete reduction using warp-level primitives
    if (idx < 32) {
        #pragma unroll
        for (int offset = 32; offset < blockDim.x; offset *= 2) {
            partial_sum += (idx + offset < blockDim.x) ? reduction_buffer[idx + offset] : 0.0f;
            reduction_buffer[idx] = partial_sum;
            __syncthreads();
        }
    }

    // Broadcast the reduced result
    const float total_sum = reduction_buffer[0];

    // Compute gradient in parallel
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[block_offset + i] = Y[block_offset + i] * (dY[block_offset + i] - total_sum);
    }
}