#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    // Optimize memory access pattern by ensuring coalesced memory access
    hiprandState localState;
    hiprand_init(seed, tidx, 0, &localState);
    state[tidx] = localState;
}