#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    // Ensure tx is within the valid range
    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        // Use efficient intrinsic function for computing logarithm
        labelLogProbs[tx] = __logf(labelp);

        // Use a ternary operator for efficiency in conditions
        int numMax = 0;
        #pragma unroll
        for (int i = 0; i < numOut; i++) {
            numMax += (probs[i * numCases + tx] == maxp);
        }
        
        // Use ternary operator for compact conditional assignment
        correctProbs[tx] = (labelp != maxp) ? 0.0f : 1.0f / float(numMax);
    }
}