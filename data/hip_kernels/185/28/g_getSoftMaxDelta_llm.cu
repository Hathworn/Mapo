#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void g_getSoftMaxDelta(float* softMaxDelta, float* softMaxP, float* groudTruth, int len)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if(id < len)
    {
        softMaxDelta[id] = softMaxP[id] - groudTruth[id]; // Compute delta only for valid index
    }
}