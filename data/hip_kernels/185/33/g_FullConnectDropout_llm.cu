#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void g_FullConnectDropout(float * outputs, float * drop, int len)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Efficient loop by striding across the data
    for (int i = id; i < len; i += blockDim.x * gridDim.x)
    {
        outputs[i] *= drop[i]; // Apply dropout
    }
}