#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void g_One_feedforward(float* _inputs, float* _w, float* _b, float* _outputs, int rows, int cols, int channels)
{
    int row     = blockIdx.x;
    int channel = blockIdx.y;

    int skip = channel * rows * cols + row * cols;
    float* inputs = _inputs + skip;
    float* outputs= _outputs + skip;
    float* w = _w + channel * cols;
    float* b = _b + channel * cols;

    // Unroll the loop for better performance
    for(int i = threadIdx.x; i < cols; i += blockDim.x){
        outputs[i] = inputs[i] * w[i] + b[i];
    }
}
```
