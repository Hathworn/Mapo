#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void g_getSoftMaxP(float* softMaxP, float* b, int cols) {
    int bid = blockIdx.x;
    extern __shared__ float _share[];
    float* _max = _share;
    float* _sum = _share + blockDim.x;
    float* sp = softMaxP + bid * cols;

    // Initialize shared memory values
    _sum[threadIdx.x] = 0.0;
    _max[threadIdx.x] = -FLT_MAX;

    // Calculate max and accumulate sum
    for (int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
        sp[tid] += b[tid];
        _max[threadIdx.x] = max(_max[threadIdx.x], sp[tid]);
    }
    __syncthreads();

    // Reduce to find global max
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            _max[threadIdx.x] = max(_max[threadIdx.x], _max[threadIdx.x + stride]);
        }
        __syncthreads();
    }

    // Normalize and exponentiate
    float max_val = _max[0];
    for (int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
        sp[tid] -= max_val;
        sp[tid] = __expf(sp[tid]);
        _sum[threadIdx.x] += sp[tid];
    }
    __syncthreads();

    // Reduce to find global sum
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            _sum[threadIdx.x] += _sum[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Final normalization
    float sum_val = _sum[0];
    for (int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
        sp[tid] /= sum_val;
    }
}