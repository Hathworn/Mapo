#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void g_One_wgrad_Add(float* _WgradTmp, float* Wgrad, float* w, int rows, int cols, int channels, float lambda)
{
    extern __shared__ float _sum[];
    int channel = blockIdx.x;
    int col     = blockIdx.y;
    int tid     = threadIdx.x;
    _sum[tid] = 0.0f;
    __syncthreads();

    // Use stride loop to reduce global memory access
    for(int i = tid; i < rows; i += blockDim.x){
        _sum[tid] += _WgradTmp[channel * rows * cols + i * cols + col];
    }
    __syncthreads();

    // Perform parallel reduction for improved performance
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
        if (tid < offset) {
            _sum[tid] += _sum[tid + offset];
        }
        __syncthreads();
    }

    if(tid == 0) {
        Wgrad[channel * cols + col] = _sum[0] / rows + w[channel * cols + col] * lambda;
    }
}