#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void g_One_backpropagation(float* _curDelta, float* _w, float* _nextDelta, int rows, int cols, int channels)
{
    int row = blockIdx.x;
    int channel = blockIdx.y;
    int tid = threadIdx.x; // Current thread index

    // Calculate skip once outside the loop
    int skip = channel * rows * cols + row * cols;
    float* curDelta = _curDelta + skip;
    float* nextDelta = _nextDelta + skip;
    float* w = _w + channel * cols;

    // Process data with shared memory to improve access efficiency
    __shared__ float shared_w[1024]; // Adjust size as needed based on `cols`
    if (tid < cols) {
        shared_w[tid] = w[tid];
    }
    __syncthreads(); // Ensure all threads have loaded weights into shared memory

    // Loop with stride to handle remaining elements
    for (int i = tid; i < cols; i += blockDim.x) {
        if (i < cols) {
            nextDelta[i] = curDelta[i] * shared_w[i];
        }
    }
}