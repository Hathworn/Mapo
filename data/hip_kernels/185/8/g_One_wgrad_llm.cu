#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void g_One_wgrad(float* _inputs, float* _curDelta, float* _wgradTmp, int rows, int cols, int channels)
{
    int row = blockIdx.x;
    int channel = blockIdx.y;
    int threadId = threadIdx.x;
    
    int skip = channel * rows * cols + row * cols;
    float* inputs = _inputs + skip;
    float* curDelta = _curDelta + skip;
    float* wgradTmp = _wgradTmp + skip;

    // Calculate global index for this thread
    int id = threadId;
    
    // Loop with stride to cover all elements in cols
    for (; id < cols; id += blockDim.x) {
        // Efficient load and computation
        wgradTmp[id] = inputs[id] * curDelta[id];
    }
}