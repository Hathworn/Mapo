#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void g_One_Bgrad(float* _delta, float* bgrad, int rows, int cols, int channels)
{
    extern __shared__ float _sum[];
    int channel = blockIdx.x;
    int col     = blockIdx.y;
    int row     = threadIdx.x;
    float delta = _delta[channel * rows * cols + row * cols + col];
    _sum[row] = delta;
    __syncthreads();

    // Reduce sum using shared memory with fewer synchronizations
    for (int stride = rows >> 1; stride > 0; stride >>= 1) {
        if (row < stride) {
            _sum[row] += _sum[row + stride];
        }
        __syncthreads();
    }

    if (row == 0) {
        bgrad[channel * cols + col] = _sum[0] / rows;
    }
}