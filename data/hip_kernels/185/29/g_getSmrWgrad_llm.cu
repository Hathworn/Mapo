#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void g_getSmrWgrad(float* wgrad, float* weight, float lambda, int len, int batch)
{
    // Calculate the global index for each thread
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop over the elements with stride equal to the total number of threads
    while (id < len) {
        // Update wgrad
        wgrad[id] = lambda * weight[id] + wgrad[id] / batch;
        
        // Move to the next set of elements
        id += gridDim.x * blockDim.x;
    }
}