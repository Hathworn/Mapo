#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void g_FullConnectWgrad(float* wgrad, float* w, int len, float lambda, int batch)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Efficiently iterate through all elements by striding over the entire grid
    for(int i = id; i < len; i += blockDim.x * gridDim.x)
    {
        if(fabs(lambda) < 1e-10)
            wgrad[i] = wgrad[i] / batch; // Simplified computation when lambda is negligible
        else
            wgrad[i] = (wgrad[i] / batch + lambda * w[i]); // Regular computation with lambda influence
    }
}