#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Use hiprand_init with unique sequence numbers for each block and thread
    hiprand_init(seed, tidx, 0, &state[tidx]);
}