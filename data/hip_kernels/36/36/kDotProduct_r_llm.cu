#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numCols, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Load data into shared memory and perform dot product operations
    if (eidx < numCols) {
        for (; eidx < numElements; eidx += numCols) {
            sum += a[eidx] * b[eidx];
        }
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Perform parallel reduction in shared memory
    if (threadIdx.x < 256) { shmem[threadIdx.x] += shmem[threadIdx.x + 256]; }
    __syncthreads();
    if (threadIdx.x < 128) { shmem[threadIdx.x] += shmem[threadIdx.x + 128]; }
    __syncthreads();
    if (threadIdx.x < 64) { shmem[threadIdx.x] += shmem[threadIdx.x + 64]; }
    __syncthreads();

    // Further reduce using warp shuffle
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
        // Write final result for this block
        if (threadIdx.x == 0) {
            target[blockIdx.x] = mysh[0];
        }
    }
}