#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Use shared memory to reduce global memory access
    __shared__ float tile[256]; // Adjust size based on block size if necessary

    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Load tile from global memory to shared memory
        tile[threadIdx.x] = src[srcY * srcWidth + srcX];
        __syncthreads();

        // Store tile back to global memory
        tgt[i] = tile[threadIdx.x];
    }
}