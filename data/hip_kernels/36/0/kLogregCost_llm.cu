#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use constant memory to store thread size
#define LOGREG_ERR_THREADS_X 256

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * blockDim.x + threadIdx.x;  // Use blockDim for thread index calculation

    if (tx < numCases) {
        const int label = (int)labels[tx];
        const float maxp = maxProbs[tx];
        const float labelp = __ldg(&probs[label * numCases + tx]);  // Use __ldg for global memory read on constant data

        labelLogProbs[tx] = __logf(labelp);

        // Use conditional operator to streamline condition check
        correctProbs[tx] = (labelp == maxp) ? 
            1.0f / float(thrust::reduce(thrust::device, probs + tx, probs + tx + numOut * numCases, 0, thrust::plus<int>())) 
            : 0.0f;
    }
}