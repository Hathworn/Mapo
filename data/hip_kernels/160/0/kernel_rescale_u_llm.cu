#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
sergeim19
April 27, 2015
Burgers equation - GPU CUDA version
*/


#define NADVANCE (4000)
#define nu (5.0e-2)

__global__ void kernel_rescale_u(double *u_dev, int N)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within the range
    if (j < N) {
        u_dev[j] /= (double)N; // Perform division inline
    }
}