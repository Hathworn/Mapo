#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NADVANCE (4000)
#define nu (5.0e-2)

__global__ void kernel_calc_uu(double *u_dev, double *uu_dev)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds to ensure we do not access memory out of range
    if(j < NADVANCE) {
        uu_dev[j] = 0.5 * u_dev[j] * u_dev[j];
    }
}