#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void get_c_size(int *d_c_size, int *d_full_cl, int size)
{
    // Calculate the global index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Check if index is within the array bounds
    if (i < size) {
        // Use atomic add only if condition is met to minimize divergence
        if (d_full_cl[i] != 0) {
            atomicAdd(d_c_size, 1);
        }
    }
}