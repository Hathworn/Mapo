#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compress_s_write_permutation(unsigned short *d_s_write_permutation, unsigned short *d_s_write_permutation_offset, int *d_write_permutation, int c_size, int chunk)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= c_size * chunk) {
        return;
    }

    // Compute chunk_id and index within the chunk
    int chunk_id = i / chunk;
    int index_within_chunk = i % chunk;

    // Load value once and use it for both assignments
    int write_perm_value = d_write_permutation[i];

    // Simplified and optimized calculations
    d_s_write_permutation[i] = (unsigned short)(write_perm_value % USHORT_MAX);

    // Only computed for the first element in each chunk
    if (index_within_chunk == 0) {
        d_s_write_permutation_offset[chunk_id] = (unsigned short)(write_perm_value / USHORT_MAX);
    }
}