#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_packed_cl_cs(int *d_packed_cl, int *d_packed_cs, int *d_cl, int *d_cs, int *d_gcs, int chunk_num)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if (i < chunk_num) {
        int gcs_diff = d_gcs[i + 1] - d_gcs[i];
        
        // Only proceed if there are entries to pack
        if (gcs_diff > 0) {
            int gcs_index = d_gcs[i];
            
            // Pack cl and cs values
            d_packed_cl[gcs_index] = d_cl[i];
            d_packed_cs[gcs_index] = d_cs[i];
        }
    }
}
```
