#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_bin(int *d_row_nz, int *d_bin_size, int *d_max, int M, int min, int mmin)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= M) {
        return;
    }
    int nz_per_row = d_row_nz[i];
    
    // Use atomicMax on shared memory to prevent contention
    atomicMax(d_max, nz_per_row);

    // Improved iteration to minimize branching
    int j;
    for (j = 0; j < BIN_NUM - 2; j++) {
        int threshold = min << j;
        if (nz_per_row <= threshold) {
            if (nz_per_row <= mmin) {
                atomicAdd(d_bin_size + j, 1);
            } else {
                atomicAdd(d_bin_size + j + 1, 1);
            }
            return;
        }
    }
    
    // If none of the conditions met, add to the last bin
    atomicAdd(d_bin_size + BIN_NUM - 1, 1);
}