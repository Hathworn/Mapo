#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compress_write_permutation(int *d_write_permutation, int *d_full_write_permutation, int *d_gcs, int total_pad_row_num, int chunk)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < total_pad_row_num) {
        int chunk_id = i / chunk;
        int start_index = d_gcs[chunk_id];
        int end_index = d_gcs[chunk_id + 1];

        // Check if there's work to do for this chunk
        if (end_index > start_index) {
            int tid = i % chunk;
            // Simplified calculation for destination index
            d_write_permutation[start_index * chunk + tid] = d_full_write_permutation[i];
        }
    }
}