#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void set_row_perm(int *d_bin_size, int *d_bin_offset, int *d_max_row_nz, int *d_row_perm, int M, int min, int mmin)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= M) {
        return;
    }

    int nz_per_row = d_max_row_nz[i];
    int dest;
    int j = 0;

    // Use shared memory to store bin_size and bin_offset to reduce global memory accesses
    __shared__ int bin_size[BIN_NUM];
    __shared__ int bin_offset[BIN_NUM];

    if (threadIdx.x < BIN_NUM) {
        bin_size[threadIdx.x] = d_bin_size[threadIdx.x];
        bin_offset[threadIdx.x] = d_bin_offset[threadIdx.x];
    }
    __syncthreads();

    // Unroll loop to reduce iterations and improve performance
    #pragma unroll
    for (j = 0; j < BIN_NUM - 2; j++) {
        if (nz_per_row <= (min << j)) {
            if (nz_per_row <= mmin) {
                dest = atomicAdd(bin_size + j, 1);
                d_row_perm[bin_offset[j] + dest] = i;
            } else {
                dest = atomicAdd(bin_size + j + 1, 1);
                d_row_perm[bin_offset[j + 1] + dest] = i;
            }
            // Synchronize bin_sizes back to global memory
            atomicExch(d_bin_size + j, bin_size[j]);
            atomicExch(d_bin_size + j + 1, bin_size[j + 1]);
            return;
        }
    }
    dest = atomicAdd(bin_size + BIN_NUM - 1, 1);
    d_row_perm[bin_offset[BIN_NUM - 1] + dest] = i;
    atomicExch(d_bin_size + BIN_NUM - 1, bin_size[BIN_NUM - 1]);
}