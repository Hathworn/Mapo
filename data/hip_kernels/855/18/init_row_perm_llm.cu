#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_row_perm(int *d_permutation, int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < M) { // Combine the two operations into one to enhance readability and performance
        d_permutation[i] = i;
    }
}