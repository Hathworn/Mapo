#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_check(int *d_check, int nz)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nz) {  // Process only valid indices
        d_check[i] = -1;
    }
}