#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_check(int *d_check, int nz)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Early exit for out-of-bounds thread
    if (i < nz) {
        d_check[i] = -1;
    }
}