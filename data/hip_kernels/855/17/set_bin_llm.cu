#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_bin(int *d_row_nz, int *d_bin_size, int *d_max, int M, int min, int mmin)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= M) {
        return;
    }
    
    int nz_per_row = d_row_nz[i];
    
    // Use a more efficient method for updating d_max
    atomicMax(d_max, nz_per_row); 
    
    // Unroll loop to reduce branching
    if (nz_per_row <= (min << 0)) {
        atomicAdd(d_bin_size + (nz_per_row <= mmin ? 0 : 1), 1);
    }
    else if (nz_per_row <= (min << 1)) {
        atomicAdd(d_bin_size + 1, 1);
    }
    else if (nz_per_row <= (min << 2)) {
        atomicAdd(d_bin_size + 2, 1);
    }
    // Add additional else-if clauses if there are more bins
    else {
        atomicAdd(d_bin_size + BIN_NUM - 1, 1);
    }
}