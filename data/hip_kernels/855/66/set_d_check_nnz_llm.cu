#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_d_check_nnz(int *d_check_nnz, int *d_nnz_num, int pad_M, int SIGMA, int sigma_block_row)
{
    // Compute global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= pad_M) {
        return;
    }

    // Pre-calculate indices used multiple times
    int idx_y_m = blockIdx.y * pad_M + i;
    int idx_y_sigma = blockIdx.y * sigma_block_row + i / SIGMA;

    // Use scalar variable for atomic operation consistency
    if (d_nnz_num[idx_y_m] > 0) {
        atomicAdd(&(d_check_nnz[idx_y_sigma]), 1); // No need for an additional variable 'a'
    }
}