#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_cs(int *d_cl, int *d_cs, int c_size, int chunk)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if index is out of bounds
    if (i >= c_size) return;

    // Initialize d_cs using ternary operator for concise code
    d_cs[i] = (i == 0) ? 0 : d_cl[i - 1] * chunk;
}