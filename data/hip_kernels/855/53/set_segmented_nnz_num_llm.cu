#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_segmented_nnz_num(int *d_rpt, int *d_col, int *d_nnz_num, int *d_group_seg, int *d_offset, size_t seg_size, size_t seg_num, int M, int pad_M, int group_num_col)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= M) {
        return;
    }

    int offset = d_rpt[i];
    int width = d_rpt[i + 1] - offset;

    // Unroll loop for better performance
    for (int j = 0; j < width; j++) {
        int index = offset + j;
        int col = d_col[index];
        int g = col / seg_size;

        // Use atomic operation for synchronization
        int nnz_index = atomicAdd(&d_nnz_num[g * pad_M + i], 1);
        d_offset[index] = nnz_index;
        d_group_seg[index] = g;
    }
}