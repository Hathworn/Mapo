#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_permutation(int *d_permutation, int M)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use ternary operator for efficient conditional assignment
    d_permutation[i] = (i < M) ? i : d_permutation[i];
}