#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_write_permutation(int *write_permutation, int *nnz_num, int total_pad_row_num, int pad_M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for better occupancy
    for (; i < total_pad_row_num; i += blockDim.x * gridDim.x) {
        write_permutation[i] -= (i / pad_M) * pad_M;
    }
}