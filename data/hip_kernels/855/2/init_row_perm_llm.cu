#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_row_perm(int *d_permutation, int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < M) { // Simplify condition check
        d_permutation[i] = i; // Direct assignment inside condition
    }
}