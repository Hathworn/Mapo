#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_bin(int *d_row_nz, int *d_bin_size, int *d_max, int M, int min, int mmin)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= M) {
        return;
    }
    int nz_per_row = d_row_nz[i];
    
    // Use a shared variable to reduce atomic operations
    __shared__ int local_max;
    if (threadIdx.x == 0) {
        local_max = 0;
    }
    __syncthreads();

    // Update local max
    atomicMax(&local_max, nz_per_row);
    __syncthreads();

    // Write back the local max to global memory
    if (threadIdx.x == 0) {
        atomicMax(d_max, local_max);
    }

    int j = 0;
    for (j = 0; j < BIN_NUM - 2; j++) {
        if (nz_per_row <= (min << j)) {
            if (nz_per_row <= mmin) {
                atomicAdd(d_bin_size + j, 1);
            } else {
                atomicAdd(d_bin_size + j + 1, 1);
            }
            return;
        }
    }
    atomicAdd(d_bin_size + BIN_NUM - 1, 1);
}