#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_cl(int *nnz_num, int *cl, int chunk, int pad_M)
{
    int c_size = pad_M / chunk;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i >= c_size) {
        return;
    }
    
    int offset = chunk * i;
    int max = 0;

    // Loop unrolling for performance improvement
    #pragma unroll
    for (int j = 0; j < chunk; j++) {
        int length = nnz_num[offset + j];
        if (length > max) {
            max = length;
        }
    }
    cl[i] = max;
}