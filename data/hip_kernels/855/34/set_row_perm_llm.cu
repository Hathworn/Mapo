#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_row_perm(int *d_bin_size, int *d_bin_offset, int *d_max_row_nz, int *d_row_perm, int M, int min, int mmin)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= M) {
        return;
    }

    int nz_per_row = d_max_row_nz[i];
    int dest;

    // Optimize loop to reduce redundant checks
    int threshold = min;
    for (int j = 0; j < BIN_NUM - 2; j++, threshold <<= 1) {
        if (nz_per_row <= threshold) {
            int offset_index = (nz_per_row <= mmin) ? j : j + 1;
            dest = atomicAdd(d_bin_size + offset_index, 1);
            d_row_perm[d_bin_offset[offset_index] + dest] = i;
            return;
        }
    }
    
    dest = atomicAdd(d_bin_size + BIN_NUM - 1, 1);
    d_row_perm[d_bin_offset[BIN_NUM - 1] + dest] = i;
}