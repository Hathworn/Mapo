#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_check(int *d_check, int nz)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for improved parallelism
    for (; i < nz; i += blockDim.x * gridDim.x)
    {
        d_check[i] = -1;
    }
}