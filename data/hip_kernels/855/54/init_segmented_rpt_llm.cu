#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_segmented_rpt(int *d_nnz_num, int *d_seg_rpt, int total_pad_row_num)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Return early if the index is out of bounds
    if (i >= total_pad_row_num) {
        return;
    }
    
    // Simplify branching structure
    d_seg_rpt[i] = (i == 0) ? 0 : d_nnz_num[i - 1];
}