#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_row_perm(int *d_bin_size, int *d_bin_offset, int *d_max_row_nz, int *d_row_perm, int M, int min, int mmin)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= M) {
        return;
    }
    
    int nz_per_row = d_max_row_nz[i];
    int dest;

    // Unroll loop for better performance
    if (nz_per_row <= mmin) {
        for (int j = 0; j < BIN_NUM - 2; j++) {
            if (nz_per_row <= (min << j)) {
                dest = atomicAdd(d_bin_size + j, 1);
                d_row_perm[d_bin_offset[j] + dest] = i;
                return;
            }
        }
    } else {
        for (int j = 0; j < BIN_NUM - 2; j++) {
            if (nz_per_row <= (min << j)) {
                dest = atomicAdd(d_bin_size + j + 1, 1);
                d_row_perm[d_bin_offset[j + 1] + dest] = i;
                return;
            }
        }
    }

    // Handle remaining cases outside the loop
    dest = atomicAdd(d_bin_size + BIN_NUM - 1, 1);
    d_row_perm[d_bin_offset[BIN_NUM - 1] + dest] = i;
}