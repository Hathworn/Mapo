#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16

__global__ static void matMultCUDA(const float* a, size_t lda, const float* b, size_t ldb, float* c, size_t ldc, int n)
{
    __shared__ float matA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float matB[BLOCK_SIZE][BLOCK_SIZE];
    const int tidc = threadIdx.x;
    const int tidr = threadIdx.y;
    const int bidc = blockIdx.x * BLOCK_SIZE;
    const int bidr = blockIdx.y * BLOCK_SIZE;

    float results = 0;
    float comp = 0;

    // Use register variables for iterators
    #pragma unroll
    for (int j = 0; j < n; j += BLOCK_SIZE) {
        // Load input tiles into shared memory
        matA[tidr][tidc] = a[(tidr + bidr) * lda + tidc + j];
        matB[tidr][tidc] = b[(tidr + j) * ldb + tidc + bidc];
        
        __syncthreads();

        // Compute the partial product
        #pragma unroll
        for (int i = 0; i < BLOCK_SIZE; i++) {
            float t;
            comp -= matA[tidr][i] * matB[i][tidc];
            t = results - comp;
            comp = (t - results) + comp;
            results = t;
        }

        __syncthreads();
    }
    
    // Write the computed value to the output matrix
    c[(tidr + bidr) * ldc + tidc + bidc] = results;
}