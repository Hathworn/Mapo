#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 10

__global__ void add(int *a, int *b, int *c)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread is within bounds
    if (tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
}