#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c, int n)
{
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we don't access out of bounds
    if (tid < n) 
    {
        c[tid] = a[tid] + b[tid];
    }
}