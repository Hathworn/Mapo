#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA kernel optimized to compute C = k * A + B for each element

__global__ void vecAdd(double *a, double *b, double *c, int n, int k)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread operates within bounds
    if (id < n) {
        c[id] = k * a[id] + b[id];
    }
}