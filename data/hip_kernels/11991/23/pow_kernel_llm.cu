#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize power kernel using efficient access pattern
__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop through every element with stride for better utilization
    for (int i = idx; i < N; i += stride) {
        Y[i * INCY] = powf(X[i * INCX], ALPHA);
    }
}