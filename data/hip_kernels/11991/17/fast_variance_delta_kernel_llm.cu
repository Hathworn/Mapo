#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    local[id] = 0;

    // Unrolling the loop to enhance performance
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            if (i < spatial) {
                local[id] += delta[index] * (x[index] - mean[filter]);
            }
        }
    }

    __syncthreads();

    // Reduction to calculate sum across threads
    if (id < 32) { // Warp unrolling, using reduction for optimization
        for (int offset = threads / 2; offset > 0; offset >>= 1) {
            local[id] += __shfl_down_sync(0xffffffff, local[id], offset);
        }
    }

    // Store result after reduction
    if (id == 0) {
        variance_delta[filter] = local[0] * -.5f * pow(variance[filter] + .000001f, -1.5f);
    }
}