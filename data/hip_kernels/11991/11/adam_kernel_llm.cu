#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation for 1D block
    if (index >= N) return;

    float B1_pow_t = pow(B1, t); // Precompute B1^t
    float B2_pow_t = pow(B2, t); // Precompute B2^t
    float sqrt_one_minus_B2_pow_t = sqrt(1.0f - B2_pow_t); // Precompute sqrt(1-B2^t)
    float one_minus_B1_pow_t = 1.0f - B1_pow_t;  // Precompute 1-B1^t
    float rate_scaled = rate * sqrt_one_minus_B2_pow_t / one_minus_B1_pow_t; // Combine scale factors
    
    x[index] -= rate_scaled * m[index] / (sqrt(v[index]) + eps); // Update x
}