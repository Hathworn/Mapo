#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use strided access pattern for coalesced memory access
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x) 
    {
        X[idx * INCX] = ALPHA;
    }
}