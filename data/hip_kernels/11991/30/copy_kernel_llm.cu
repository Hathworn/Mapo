#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY) {
    // Calculate unique thread's global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;  // Calculate grid stride

    // Loop through array with grid stride loop to handle more elements
    for(int j = i; j < N; j += stride) {
        Y[j * INCY + OFFY] = X[j * INCX + OFFX];  // Perform copy operation
    }
}