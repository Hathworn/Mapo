#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Use shared memory for frequently accessed data
    extern __shared__ float shared_mem[];
    if (threadIdx.x < n) {
        shared_mem[threadIdx.x] = dc[threadIdx.x];
        __syncthreads(); // Ensure all threads have loaded data
    }
    
    if (i < n) {
        float dc_val = shared_mem[i % blockDim.x]; // Use shared memory data
        float si = s[i];
        
        if (da) da[i] += dc_val * si; // Update da using shared value
        db[i] += dc_val * (1 - si); // Update db using shared value
        ds[i] += dc_val * a[i] - dc_val * b[i]; // Update ds using shared value
    }
}