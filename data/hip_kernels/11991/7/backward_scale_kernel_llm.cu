#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCK 1024

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];  
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    for(int b = 0; b < batch; ++b) {
        for(int i = p; i < size; i += BLOCK) {  // Parallelize across threads
            int index = i + size * (filter + n * b);
            sum += delta[index] * x_norm[index];
        }
    }
    
    part[p] = sum;
    __syncthreads();

    if (p == 0) {
        float block_sum = 0;
        for(int i = 0; i < BLOCK; ++i) block_sum += part[i];  // Reduce within block
        atomicAdd(&scale_updates[filter], block_sum);  // Atomic add to global memory
    }
}