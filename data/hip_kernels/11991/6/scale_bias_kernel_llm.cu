#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    // Use shared memory to reduce global memory accesses
    __shared__ float shared_biases[1024]; // Assuming max thread block size of 1024

    int tid = threadIdx.x;
    int offset = blockIdx.x * blockDim.x + tid;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Load bias into shared memory
    if (tid == 0) {
        shared_biases[filter] = biases[filter];
    }
    __syncthreads();

    // Ensure offset is within bounds
    if (offset < size) {
        output[(batch * n + filter) * size + offset] *= shared_biases[filter];
    }
}