#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Unroll the inner loop to improve efficiency
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; i += BLOCK * 4) {
            int idx = p + i + size * (filter + n * b);
            sum += (p + i < size) ? delta[idx] : 0;
            sum += (p + i + BLOCK < size) ? delta[idx + BLOCK] : 0;
            sum += (p + i + 2 * BLOCK < size) ? delta[idx + 2 * BLOCK] : 0;
            sum += (p + i + 3 * BLOCK < size) ? delta[idx + 3 * BLOCK] : 0;
        }
    }

    part[p] = sum;
    __syncthreads();

    // Use more efficient reduction within the block
    if (p < 32) {
        for (int i = BLOCK / 2; i >= 32; i >>= 1) {
            if (p < i) part[p] += part[p + i];
            __syncthreads();
        }
        if (p < 16) part[p] += part[p + 16];
        if (p < 8) part[p] += part[p + 8];
        if (p < 4) part[p] += part[p + 4];
        if (p < 2) part[p] += part[p + 2];
        if (p == 0) bias_updates[filter] += part[0] + part[1];
    }
}