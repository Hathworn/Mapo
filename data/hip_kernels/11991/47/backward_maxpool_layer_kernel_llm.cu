#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + 2*pad) / stride;
    int w = (in_w + 2*pad) / stride;
    int c = in_c;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    int w_offset = -pad;
    int h_offset = -pad;

    float d = 0;
    int area = (size - 1) / stride;
    // Unrolling loop for performance improvement
    #pragma unroll
    for (int l = -area; l <= area; ++l) {
        #pragma unroll
        for (int m = -area; m <= area; ++m) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;
            int out_index = out_w + w * (out_h + h * (k + c * b));
            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h) {
                if (indexes[out_index] == id) {
                    d += delta[out_index];    
                }
            }
        }
    }
    atomicAdd(&prev_delta[b * in_c * in_h * in_w + k * in_h * in_w + i * in_w + j], d);
}