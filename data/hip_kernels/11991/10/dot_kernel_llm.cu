#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Compute thread index
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0.0f;
    float norm1 = 0.0f;
    float norm2 = 0.0f;
    
    // Use register memory for better performance in loops
    #pragma unroll
    for (int b = 0; b < batch; ++b) {
        #pragma unroll
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            float o1 = output[i1];
            float o2 = output[i2];
            sum += o1 * o2;
            norm1 += o1 * o1;
            norm2 += o2 * o2;
        }
    }
    norm1 = sqrtf(norm1);
    norm2 = sqrtf(norm2);
    float norm = norm1 * norm2;
    sum /= norm;

    // Pre-calculate scaled sum over norm
    float factor = -scale * sum / norm;

    #pragma unroll
    for (int b = 0; b < batch; ++b) {
        #pragma unroll
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            float o1 = output[i1];
            float o2 = output[i2];
            delta[i1] += factor * o2;
            delta[i2] += factor * o1;
        }
    }
}