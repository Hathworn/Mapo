#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel
__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    // Calculate the thread's global index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    int b = id / c;

    int out_index = k + c * b;
    float sum = 0;  // Use register to store sum

    // Unroll the loop for better performance
    for(int i = 0; i < w * h; ++i){
        int in_index = i + h * w * (k + b * c);
        sum += input[in_index];
    }
    output[out_index] = sum / (w * h);
}