#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    // Use stride to process all elements with fewer threads
    for (; i < n; i += blockDim.x * gridDim.x * gridDim.y) {
        if (mask[i] == mask_num) {
            x[i] = mask_num;
        }
    }
}