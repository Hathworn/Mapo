#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use a 1D index for optimizing global memory access pattern
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < N) {
        // Pre-compute ALPHA^2 for reduced operations inside the loop
        float alphaSquared = ALPHA * ALPHA;
        float val = X[idx * INCX];
        
        // Optimization: use a temporary variable to store squared value
        float valSquared = val * val;

        if(valSquared < alphaSquared) {
            X[idx * INCX] = 0;
        }
    }
}