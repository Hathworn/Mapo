#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Improve memory access pattern by using shared memory for biases
    __shared__ float shared_biases[512];
    if (threadIdx.x == 0) {
        shared_biases[filter] = biases[filter];
    }
    __syncthreads();
    
    if(offset < size) {
        output[(batch*n+filter)*size + offset] += shared_biases[filter];
    }
}