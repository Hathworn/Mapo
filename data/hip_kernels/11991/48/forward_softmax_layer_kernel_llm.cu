#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forward_softmax_layer_kernel(int n, int batch, float *input, float temp, float *output)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if(b >= batch) return;

    // Find the largest element to improve numerical stability
    float sum = 0.0f;
    float largest = -INFINITY;
    for(int i = 0; i < n; ++i){
        largest = max(largest, input[i + b * n]);
    }

    // Compute the sum of exponentials
    for(int i = 0; i < n; ++i){
        sum += expf((input[i + b * n] - largest) / temp);
    }

    // Normalize the output with the calculated sum
    float expSum = largest + temp * logf(sum);
    for(int i = 0; i < n; ++i){
        output[i + b * n] = expf((input[i + b * n] - expSum) / temp);
    }
}