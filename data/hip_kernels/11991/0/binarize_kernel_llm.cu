#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Calculate global thread index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    if (i < n) {
        binary[i] = (x[i] >= 0) ? 1 : -1;
    }
}