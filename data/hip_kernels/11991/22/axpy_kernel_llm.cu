#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Use blockIdx.x + blockIdx.y*gridDim.x to flatten and calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Check if the thread index i is within bounds, then compute
    if (i < N) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}