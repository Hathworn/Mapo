#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    float sum = 0.0f;  // Temporary variable to accumulate results

    for(int j = 0; j < batch; ++j){
        for(int i = id; i < spatial; i += threads){ // Spread work evenly across threads
            int index = j * spatial * filters + filter * spatial + i;
            sum += delta[index]; // Accumulate directly to avoid bank conflicts
        }
    }

    local[id] = sum; // Store the partial sum in shared memory
    __syncthreads(); // Synchronize threads to ensure all sums are written

    // Reduce the sum in shared memory to a single value
    if (id == 0) {
        float total_sum = 0.0f;
        for(int i = 0; i < threads; ++i) {
            total_sum += local[i];
        }
        mean_delta[filter] = total_sum * (-1.0f / sqrtf(variance[filter] + 0.000001f));
    }
}