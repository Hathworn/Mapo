#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index considering INCX
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i * INCX < N) 
    {
        X[i * INCX] = ALPHA;  // Assign ALPHA to the calculated position
    }
}