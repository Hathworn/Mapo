#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel to constrain values
__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index for the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // If the index is valid
    if (i < N) {
        // Compute the effective index considering the increment
        int idx = i * INCX;
        // Apply constraints to the value in X
        X[idx] = fminf(ALPHA, fmaxf(-ALPHA, X[idx]));
    }
}