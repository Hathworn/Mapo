#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation to 1D
    if (i < n) {
        float bi = (b ? b[i] : 0); // Avoid conditional evaluation inside calculation
        c[i] = s[i] * a[i] + (1 - s[i]) * bi;
    }
}