#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    int stride = blockDim.x * gridDim.x; // Calculate grid stride
    for (; id < size; id += stride) { // Loop with grid stride
        input[id] = (rand[id] < prob) ? 0 : input[id] * scale; // Perform conditional operation
    }
}