#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    // Calculate the global index
    int s = blockIdx.x * blockDim.x + threadIdx.x;
    if (s >= size) return;

    // Use shared memory to reduce repetitive global memory access
    extern __shared__ float shared_input[];

    int i = 0;
    float mean = 0;

    // Load data into shared memory
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        shared_input[i] = abs(input[i * size + s]);
    }
    __syncthreads();

    // Calculate mean
    for (i = 0; i < n; ++i) {
        mean += shared_input[i];
    }
    mean = mean / n;

    // Binarize input
    for (i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}