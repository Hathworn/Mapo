#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    extern __shared__ float shared_data[]; // Use shared memory
    float sum = 0.0f;
    float largest = -INFINITY;

    // Use one loop with parallel reduction to find largest
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        float val = input[i];
        atomicMax(&largest, val); // Use atomic operation to find largest in parallel
    }
    __syncthreads();

    // Compute exponentials and sum them up
    float partial_sum = 0.0f;
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        float e = exp((input[i] - largest) / temp);
        partial_sum += e;
        output[i] = e;
    }
    shared_data[threadIdx.x] = partial_sum; // Store partial sum in shared memory
    __syncthreads();

    // Reduce partial sums into total sum
    if (threadIdx.x == 0) {
        for (int i = 1; i < blockDim.x; ++i) {
            sum += shared_data[i];
        }
        sum += shared_data[0]; // Add the first element
    }
    __syncthreads();

    // Normalize output
    float norm_factor = sum; // Ensures the entire block sees the sum
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        output[i] /= norm_factor;
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x; // Simplify grid index calculation
    if (b >= batch) return;
    softmax_device(n, input + b * offset, temp, output + b * offset);
}