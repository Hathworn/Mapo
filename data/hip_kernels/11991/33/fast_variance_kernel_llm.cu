#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0;

    // Use loop unrolling for improved efficiency
    for (int j = 0; j < batch; ++j) {
        int baseIndex = j * spatial * filters + filter * spatial;
        for (int i = id; i < spatial; i += threads) {
            int index = baseIndex + i;
            local[id] += pow((x[index] - mean[filter]), 2);
        }
    }

    // Use __syncthreads() for synchronization
    __syncthreads();

    // Use parallel reduction for improved performance
    for (int offset = threads / 2; offset > 0; offset >>= 1) {
        if (id < offset) {
            local[id] += local[id + offset];
        }
        __syncthreads();
    }

    // Output final result
    if (id == 0) {
        variance[filter] = local[0] / (spatial * batch - 1);
    }
}