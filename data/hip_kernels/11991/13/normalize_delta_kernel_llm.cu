#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    // Calculate unique thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure indexing within bounds
    if (index >= N) return;
    
    // Calculate the filter index
    int f = (index / spatial) % filters;
    
    // Pre-calculate commonly used values
    float inv_denom = 1.0f / (sqrt(variance[f]) + 0.000001f);
    float multiplier = 2.0f / (spatial * batch);

    // Compute new delta with pre-calculated values
    delta[index] = delta[index] * inv_denom + 
                   variance_delta[f] * multiplier * (x[index] - mean[f]) + 
                   mean_delta[f] / (spatial * batch);
}