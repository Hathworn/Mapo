#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float acc = 0.0f;  // Accumulator for variance_delta
    for(int j = 0; j < batch; ++j) {
        for(int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            acc += delta[index] * (x[index] - mean[i]);
        }
    }

    // Use intrinsic rsqrtf for fast inverse square root
    variance_delta[i] = acc * -0.5f * rsqrtf((variance[i] + .000001f) * (variance[i] + .000001f) * (variance[i] + .000001f) );
}