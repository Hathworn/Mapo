#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use a stride loop to cover all elements
    for (; i < N; i += stride)
    {
        Y[i * INCY] *= X[i * INCX];
    }
}