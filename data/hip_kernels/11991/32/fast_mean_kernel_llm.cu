#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Unrolling spatial loop by a factor of 4 to enhance performance
    int i, j;
    for (j = 0; j < batch; ++j) {
        for (i = 0; i < spatial; i += threads * 4) {
            int base_index = j * spatial * filters + filter * spatial + i + id;
            float sum = 0;
            if (i + id < spatial) sum += x[base_index];
            if (i + threads + id < spatial) sum += x[base_index + threads];
            if (i + 2 * threads + id < spatial) sum += x[base_index + 2 * threads];
            if (i + 3 * threads + id < spatial) sum += x[base_index + 3 * threads];
            local[id] += sum;
        }
    }

    __syncthreads(); // Ensure all threads have computed their local sums

    // Reduce within block to compute the mean
    if (id == 0) {
        float sum = 0;
        for (i = 0; i < threads; ++i) {
            sum += local[i];
        }
        mean[filter] = sum / (spatial * batch);
    }
}