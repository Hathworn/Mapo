#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;

    float mean = 0;
    // Use shared memory for mean calculation to reduce global memory accesses
    __shared__ float shared_weights[1024]; // Assuming a max block size of 1024

    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[i] = abs(weights[f * size + i]);
    }
    __syncthreads(); // Ensure all shared memory loads are complete

    // Parallel reduction to compute mean
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s && (threadIdx.x + s) < size) {
            shared_weights[threadIdx.x] += shared_weights[threadIdx.x + s];
        }
        __syncthreads();
    }

    if(threadIdx.x == 0) {
        mean = shared_weights[0] / size;
    }
    __syncthreads(); // Ensure all threads get the computed mean

    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    }
}