#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    // Initialize local sum for each thread
    float local_sum = 0.0f;

    // Use local accumulation before writing to global memory to minimize global memory accesses
    for (int k = 0; k < n; ++k)
    {
        local_sum += x[k * groups + i];
    }

    // Write the local sum to global memory
    sum[i] = local_sum;
}