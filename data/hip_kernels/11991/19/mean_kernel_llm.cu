#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.0f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation
    if (i >= filters) return;

    extern __shared__ float shared_mean[];  // Shared memory to accumulate mean
    shared_mean[threadIdx.x] = 0.0f;

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            shared_mean[threadIdx.x] += x[index];
        }
    }

    // Reduce in shared memory
    __syncthreads();

    if (threadIdx.x == 0) {
        float block_sum = 0.0f;
        for (int s = 0; s < blockDim.x; ++s) {
            block_sum += shared_mean[s];
        }
        atomicAdd(&mean[i], block_sum * scale);  // Use atomicAdd to accumulate results
    }
}