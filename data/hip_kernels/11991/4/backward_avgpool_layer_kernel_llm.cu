#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    // Calculate the global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Safeguard condition: only consider valid indices
    if (id >= n * c * w * h) return;

    // Optimize indexing
    // Compute batch, channel, and spatial indices
    int spatial_index = id % (w * h);
    int k = (id / (w * h)) % c;
    int b = id / (c * w * h);

    // Compute indices for input and output deltas
    int in_index = spatial_index + w * h * (k + b * c);
    int out_index = k + c * b;

    // Update input delta
    in_delta[in_index] += out_delta[out_index] / (w * h);
}