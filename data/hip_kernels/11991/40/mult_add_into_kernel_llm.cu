#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Optimize index calculation by avoiding repetitive calculations
    int idx = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    if(idx < n) {
        c[idx] += a[idx] * b[idx];
    }
}