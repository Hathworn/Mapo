#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    float h = hsv.x;
    float s = hsv.y;
    float v = hsv.z;

    float r, g, b;
    float f, p, q, t;

    // Simplified condition for when s is zero
    if (s == 0) {
        r = g = b = v;
    } else {
        int index = (int) floorf(h);
        f = h - index;
        p = v * (1 - s);
        q = v * (1 - s * f);
        t = v * (1 - s * (1 - f));
        
        // Simplified branching with a small array lookup table
        float3 rgb[6] = {
            make_float3(v, t, p), make_float3(q, v, p),
            make_float3(p, v, t), make_float3(p, q, v),
            make_float3(t, p, v), make_float3(v, p, q)
        };
        r = rgb[index].x;
        g = rgb[index].y;
        b = rgb[index].z;
    }
    // Clamp values between 0 and 1
    r = fminf(fmaxf(r, 0.0f), 1.0f);
    g = fminf(fmaxf(g, 0.0f), 1.0f);
    b = fminf(fmaxf(b, 0.0f), 1.0f);
    return make_float3(r, g, b);
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    float r = rgb.x;
    float g = rgb.y;
    float b = rgb.z;

    float h, s, v;
    float max = fmaxf(r, fmaxf(g, b));
    float min = fminf(r, fminf(g, b));
    float delta = max - min;
    v = max;

    if (max == 0) {
        s = 0;
        h = -1;
    } else {
        s = delta / max;
        if (r == max) {
            h = (g - b) / delta;
        } else if (g == max) {
            h = 2 + (b - r) / delta;
        } else {
            h = 4 + (r - g) / delta;
        }
        // Wrap hue to range [0,6]
        if (h < 0) h += 6;
    }
    return make_float3(h, s, v);
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int size = batch * w * h;
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    
    // Exit early if out of bounds
    if (id >= size) return;

    int x = id % w;
    id /= w;
    int y = id % h;
    id /= h;

    // Use shared memory for random values
    __shared__ float shared_rand[3];
    if (threadIdx.x < 3) shared_rand[threadIdx.x] = rand[threadIdx.x];
    __syncthreads();

    float r0 = rand[8 * id + 0];
    float r1 = rand[8 * id + 1];
    float r2 = rand[8 * id + 2];
    float r3 = rand[8 * id + 3];

    saturation = r0 * (saturation - 1) + 1;
    saturation = (r1 > .5) ? 1. / saturation : saturation;
    exposure = r2 * (exposure - 1) + 1;
    exposure = (r3 > .5) ? 1. / exposure : exposure;

    size_t offset = id * h * w * 3;
    image += offset;
    float r = image[x + w * (y + h * 0)];
    float g = image[x + w * (y + h * 1)];
    float b = image[x + w * (y + h * 2)];
    float3 rgb = make_float3(r, g, b);

    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }
    // Update image with calculated values
    image[x + w * (y + h * 0)] = rgb.x * scale + translate + (shared_rand[0] - .5) * shift;
    image[x + w * (y + h * 1)] = rgb.y * scale + translate + (shared_rand[1] - .5) * shift;
    image[x + w * (y + h * 2)] = rgb.z * scale + translate + (shared_rand[2] - .5) * shift;
}