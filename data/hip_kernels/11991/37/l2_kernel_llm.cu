#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (i < n) {
        float diff = truth[i] - pred[i];
        error[i] = diff * diff; // Compute error as square of the difference
        delta[i] = diff; // Store difference in delta
    }
}