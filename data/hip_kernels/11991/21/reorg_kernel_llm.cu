#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (idx >= N) return; // Ensure we don't exceed the array bounds

    // Decompose the index into width, height, channel, and batch
    int in_w = idx % w;
    int in_h = (idx / w) % h;
    int in_c = (idx / (w * h)) % c;
    int b = idx / (w * h * c);

    // Calculate the output channel and offsets
    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;

    // Calculate output index
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    // Perform the reorganization
    if (forward) out[out_index] = x[idx];
    else out[idx] = x[out_index];
}