#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread operates on valid data
    if (i >= n) return;

    // Compute difference and absolute value
    float diff = truth[i] - pred[i];
    float abs_val = abs(diff);

    // Conditionally compute error and delta based on diff
    if(abs_val < 1) {
        error[i] = diff * diff;
        delta[i] = diff;
    } else {
        error[i] = 2 * abs_val - 1;
        delta[i] = copysign(1.0f, diff); // Use copysign to efficiently assign -1 or 1
    }
}