#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) { // Use branch reduction by checking only if within bounds
        // Calculate input spatial, channel, and batch indices
        int in_s = i % spatial;
        int in_c = (i / spatial) % layers;
        int b = i / (spatial * layers);
        
        // Calculate flattened indices for input and output
        int i1 = b * layers * spatial + in_c * spatial + in_s;
        int i2 = b * layers * spatial + in_s * layers + in_c;

        // Conditional operation for forward or backward
        out[forward ? i2 : i1] = x[forward ? i1 : i2];
    }
}