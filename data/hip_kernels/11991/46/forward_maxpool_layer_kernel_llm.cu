#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    // Calculate grid dimensions
    int h = (in_h + 2*pad) / stride;
    int w = (in_w + 2*pad) / stride;
    int c = in_c;

    // Compute unique ID for each thread
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Calculate positions within feature map
    int out_index = id;
    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -pad;
    int h_offset = -pad;

    // Initialize variables for max pooling logic
    float max = -INFINITY;
    int max_i = -1;

    // Loop through pooling window
    for (int l = 0; l < size; ++l) {
        for (int m = 0; m < size; ++m) {
            int cur_h = h_offset + i * stride + l;
            int cur_w = w_offset + j * stride + m;

            // Check boundary conditions
            if (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w) {
                
                // Calculate input index within the current batch and channel
                int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));

                // Read input value and update max pooling result
                float val = input[index];
                if (val > max) {
                    max = val;
                    max_i = index;
                }
            }
        }
    }

    // Write the result to output and index arrays
    output[out_index] = max;
    indexes[out_index] = max_i;
}