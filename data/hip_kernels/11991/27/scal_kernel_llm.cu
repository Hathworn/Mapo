#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = gridDim.x * blockDim.x;

    // Loop over all elements assigned to each thread
    while (i < N) {
        X[i * INCX] *= ALPHA;
        i += gridSize; // Move to the next element the thread will process
    }
}