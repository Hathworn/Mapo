#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < filters) {
        float sum_delta = 0.0f; // Use a register for sum accumulation
        for (int j = 0; j < batch; ++j) {
            for (int k = 0; k < spatial; ++k) {
                int index = j * filters * spatial + i * spatial + k;
                sum_delta += delta[index];
            }
        }
        mean_delta[i] = sum_delta * (-1.0f / sqrtf(variance[i] + 0.000001f)); // Use sqrtf for performance
    }
}