#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified computation of index
    int gridSize = gridDim.x * blockDim.x; // Compute grid size

    while (index < N) { // Loop for processing elements in same thread
        int f = (index / spatial) % filters;

        x[index] = (x[index] - mean[f]) / (sqrtf(variance[f]) + .000001f); // Use sqrtf for better precision
        index += gridSize; // Move to the next element this thread should process
    }
}