#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void pw_copy_cr_cu_z(const double *zin, double *dout, const int n) {
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index computation

    if (igpt < n) {
        dout[igpt] = zin[2 * igpt];
    }
}