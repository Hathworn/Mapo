#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_copy_rc_cu_z(const double *din, double *zout, const int n) {
    // Calculate unique index of the thread
    int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Check if the thread is within bounds and update zout
    if (igpt < n) {
        zout[igpt * 2] = din[igpt];
        zout[igpt * 2 + 1] = 0.0;
    }
}