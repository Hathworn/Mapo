#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_gather_cu_z(double *pwcc, const double *c, const double scale, const int ngpts, const int *ghatmap) {
    // Calculate global index
    int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Check for boundary and update pwcc
    if (igpt < ngpts) {
        int idx = 2 * ghatmap[igpt]; // Reduce redundant multiplication
        pwcc[2 * igpt] = scale * c[idx];
        pwcc[2 * igpt + 1] = scale * c[idx + 1];
    }
}