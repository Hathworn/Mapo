#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_scatter_cu_z(double *c, const double *pwcc, const double scale, const int ngpts, const int nmaps, const int *ghatmap) {

    int igpt = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    
    if (igpt < ngpts) {
        int idx = 2 * ghatmap[igpt];  // Pre-compute index for memory access optimization
        int pwcc_idx = 2 * igpt;      // Pre-compute index for source array
        c[idx] = scale * pwcc[pwcc_idx];
        c[idx + 1] = scale * pwcc[pwcc_idx + 1];

        if (nmaps == 2) {
            int idx2 = 2 * ghatmap[igpt + ngpts];
            c[idx2] = scale * pwcc[pwcc_idx];
            c[idx2 + 1] = -scale * pwcc[pwcc_idx + 1];
        }
    }
}