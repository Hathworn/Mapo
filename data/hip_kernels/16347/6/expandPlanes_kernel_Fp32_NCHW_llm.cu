#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void expandPlanes_kernel_Fp32_NCHW(float* output, const uint64_t* masks, const float* values, int n) {
    // Block size of 256, same mask/val for 64 consecutive threads.
    constexpr int kNumShmemElements = 256 / 64;

    __shared__ uint64_t shMasks[kNumShmemElements];
    __shared__ float shVals[kNumShmemElements];

    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int planeIndex = index >> 6;

    if (planeIndex >= n) return;

    // Load inputs to shared memory using fewer branches.
    if (threadIdx.x < kNumShmemElements) {
        int sharedIdx = threadIdx.x;
        shMasks[sharedIdx] = masks[planeIndex + sharedIdx];
        shVals[sharedIdx] = values[planeIndex + sharedIdx];
    }
    __syncthreads();

    uint64_t mask = shMasks[threadIdx.x >> 6];
    int sqIndex = index & 0x3F;
    float op = 0;

    // Use conditional assignment to avoid branches.
    op = (mask & (1ull << sqIndex)) ? shVals[threadIdx.x >> 6] : 0;
    output[index] = op;
}