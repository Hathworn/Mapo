#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 32
#define TILE_HEIGHT 32
#define FSize 256
//void convolution(int *InputImage,int width,int height,int *filter,int filterWidth,,int padding,int *result);
using namespace std;

__global__ void shareMatrixMultiple(int *InputImage,int width,int height,int *filter,int filterWidth,int *featureMap)
{
    extern __shared__ int tileImage[];

    int Row = blockIdx.y * TILE_HEIGHT + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int shareWidth = TILE_WIDTH + filterWidth - 1;
    int value = 0;
    int featureMapWidth = width - filterWidth + 1;

    // Load tile into shared memory with boundary checks
    if (Row < height && Col < width)
        tileImage[threadIdx.y * shareWidth + threadIdx.x] = InputImage[Row * width + Col];
    
    if (threadIdx.x < filterWidth - 1 && Col + TILE_WIDTH < width)
        tileImage[threadIdx.y * shareWidth + threadIdx.x + TILE_WIDTH] = InputImage[Row * width + Col + TILE_WIDTH];
    
    if (threadIdx.y < filterWidth - 1 && Row + TILE_HEIGHT < height)
        tileImage[(threadIdx.y + TILE_HEIGHT) * shareWidth + threadIdx.x] = InputImage[(Row + TILE_HEIGHT) * width + Col];
    
    if (threadIdx.x < filterWidth - 1 && threadIdx.y < filterWidth - 1 
        && Col + TILE_WIDTH < width && Row + TILE_HEIGHT < height)
        tileImage[(threadIdx.y + TILE_HEIGHT) * shareWidth + threadIdx.x + TILE_WIDTH] = InputImage[(Row + TILE_HEIGHT) * width + Col + TILE_WIDTH];
    
    __syncthreads();

    // Perform convolution only within valid output bounds
    if(Row < featureMapWidth && Col < featureMapWidth)
    {
        for(int i = 0; i < filterWidth; i++)
        {
            for(int j = 0; j < filterWidth; j++)
            {
                value += filter[i * filterWidth + j] * tileImage[(threadIdx.y + i) * shareWidth + threadIdx.x + j];
            }
        }
        featureMap[featureMapWidth * Row + Col] = value;
    }
}