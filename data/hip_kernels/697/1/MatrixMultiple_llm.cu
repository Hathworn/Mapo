#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 32
#define TILE_HEIGHT 32
#define FSize 256
//void convolution(int *InputImage,int width,int height,int *filter,int filterWidth,,int padding,int *result);
using namespace std;

__global__ void MatrixMultiple(int *InputImage, int width, int height, int *filter, int filterWidth, int *featureMap)
{
    // Calculate global row and column index
    int Row = blockIdx.y * TILE_HEIGHT + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int value = 0;
    int featureMapWidth = width - filterWidth + 1;

    // Check if within boundaries
    if (Row < height - filterWidth + 1 && Col < width - filterWidth + 1)
    {
        // Convolution operation
        for (int i = 0; i < filterWidth; i++)
        {
            for (int j = 0; j < filterWidth; j++)
            {
                // Accumulate the values using the filter
                value += filter[i * filterWidth + j] * InputImage[(Row + i) * width + (Col + j)];
            }
        }
        // Assign computed value to featureMap
        featureMap[featureMapWidth * Row + Col] = value;
    }
}