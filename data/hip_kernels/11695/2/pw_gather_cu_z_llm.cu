#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_gather_cu_z(double *pwcc, const double *c, const double scale, const int ngpts, const int *ghatmap) {
    // Calculate global thread index
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    if (igpt < ngpts) {
        // Memory coalescing with reduced index recomputation 
        const int map_idx = 2 * ghatmap[igpt];
        const int out_idx = 2 * igpt;
        pwcc[out_idx] = scale * c[map_idx];
        pwcc[out_idx + 1] = scale * c[map_idx + 1];
    }
}