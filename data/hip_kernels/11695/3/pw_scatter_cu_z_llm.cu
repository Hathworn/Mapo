#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_scatter_cu_z(double *c, const double *pwcc, const double scale, const int ngpts, const int nmaps, const int *ghatmap) {

    // Calculate the global index for the current thread
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x + (gridDim.x * blockIdx.y);

    // Process only valid indices
    if (igpt < ngpts) {
        // Calculate the index for ghatmap once to avoid redundant calculations
        int map_idx = 2 * ghatmap[igpt];

        // Update the output array with scaling
        c[map_idx] = scale * pwcc[2 * igpt];
        c[map_idx + 1] = scale * pwcc[2 * igpt + 1];

        // Handle additional mapping if required
        if (nmaps == 2) {
            int map_idx_ngpts = 2 * ghatmap[igpt + ngpts];
            c[map_idx_ngpts] = scale * pwcc[2 * igpt];
            c[map_idx_ngpts + 1] = -scale * pwcc[2 * igpt + 1];
        }
    }
}