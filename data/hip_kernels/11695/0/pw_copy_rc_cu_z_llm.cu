#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_copy_rc_cu_z(const double *din, double *zout, const int n) {
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Optimize memory access pattern for coalesced access
    if (igpt < n) {
        zout[igpt * 2] = din[igpt];
        zout[igpt * 2 + 1] = 0.0;
    }
}