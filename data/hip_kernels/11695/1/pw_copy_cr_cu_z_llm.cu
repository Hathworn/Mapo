#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_copy_cr_cu_z(const double *zin, double *dout, const int n) {
    // Calculate the global thread ID more concisely
    int igpt = blockIdx.x * blockDim.x + threadIdx.x;

    // Utilize grid-stride loop for better performance on larger data sizes
    for (int i = igpt; i < n; i += blockDim.x * gridDim.x) {
        dout[i] = zin[2 * i]; // Efficient memory access pattern
    }
}