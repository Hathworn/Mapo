#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int greatest_row;  

__device__ void swap(float* arr, int ind_a, int ind_b) {
    float tmp = arr[ind_a];
    arr[ind_a] = arr[ind_b];
    arr[ind_b] = tmp;
}

__global__ void swapRow(float* mat, float* b, float* column_k, int rows, int cols, int k) {
    int row_i = greatest_row;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if swap is necessary and within bounds
    if (k != row_i && i <= cols) {
        if (i < cols) {
            // Swap matrix rows
            float tmp = mat[k * cols + i];
            mat[k * cols + i] = mat[row_i * cols + i];
            mat[row_i * cols + i] = tmp;
        } else {
            // Swap vector b
            float tmp = b[k];
            b[k] = b[row_i];
            b[row_i] = tmp;
        }
    }

    // Store column k only if within row bounds
    if (i < rows) {
        column_k[i] = mat[i * cols + k];
    }
}