#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sortKernelSimple(int *arr, int arr_len, int odd)
{
    // Optimize by using shared memory to reduce global memory access
    extern __shared__ int shared[];
    int i = 2 * (blockIdx.x * blockDim.x + threadIdx.x) + odd;

    if (i < arr_len - 1)
    {
        // Load data into shared memory
        shared[threadIdx.x * 2] = arr[i];
        shared[threadIdx.x * 2 + 1] = arr[i + 1];
        __syncthreads();

        // Sort the loaded elements
        int a = shared[threadIdx.x * 2];
        int b = shared[threadIdx.x * 2 + 1];
        if (a > b)
        {
            shared[threadIdx.x * 2] = b;
            shared[threadIdx.x * 2 + 1] = a;
        }
        __syncthreads();

        // Write back the sorted elements to global memory
        arr[i] = shared[threadIdx.x * 2];
        arr[i + 1] = shared[threadIdx.x * 2 + 1];
    }
}