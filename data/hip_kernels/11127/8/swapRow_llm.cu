#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int greatest_row;  

__device__ void swap(float* arr, int ind_a, int ind_b) {
    float tmp = arr[ind_a];
    arr[ind_a] = arr[ind_b];
    arr[ind_b] = tmp;
}

__global__ void swapRow(float* mat, float* b, int cols, int num_block, int k) {
    int row_i = greatest_row;
    if (k != row_i) { // If the same row don't swap.
        int row_k = k * cols;
        int swap_row = row_i * cols;
        int i = threadIdx.x + blockIdx.x * blockDim.x;

        // Optimized loop with shared memory
        extern __shared__ float shared_mem[];
        for (int j = i; j < cols; j += num_block * blockDim.x) {
            shared_mem[threadIdx.x] = mat[swap_row + j];
            __syncthreads(); // Synchronize threads
            mat[swap_row + j] = mat[row_k + j];
            mat[row_k + j] = shared_mem[threadIdx.x];
            __syncthreads(); // Synchronize threads
        }
        
        // Swap b
        if (blockIdx.x == 0 && threadIdx.x == 0)
            swap(b, row_i, k);
    }
}