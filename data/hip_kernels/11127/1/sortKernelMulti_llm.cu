#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sortKernelMulti(int *arr, int arr_len, int num_elem, int oddEven)
{
    int i = 2 * (blockIdx.x * blockDim.x * num_elem) + oddEven + 2 * threadIdx.x;
    int iterEnd = min(arr_len - 1, i + 2 * blockDim.x * num_elem);

    // Optimize by unrolling the loop to reduce branch overhead and improve instruction throughput
    #pragma unroll
    for (; i < iterEnd; i += 2 * blockDim.x)
    {
        int a = arr[i];
        int b = arr[i + 1];
        if (a > b)
        {
            arr[i] = b;
            arr[i + 1] = a;
        }
    }
}