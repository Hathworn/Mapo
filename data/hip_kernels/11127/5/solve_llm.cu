#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void solve(float* mat, float* b, float* x, int rows, int cols)
{
    int n = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim.x for flexibility
    if (n < rows) // Ensure bounds
    {
        float diag_val = mat[n * cols + n]; // Reuse memory access
        if (diag_val != 0) // Check for division by zero
        {
            x[n] = b[n] / diag_val;
        }
    }
}