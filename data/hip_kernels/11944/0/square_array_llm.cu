#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory for faster access
    extern __shared__ float s_data[];
    // Load data into shared memory
    if (idx < N) s_data[threadIdx.x] = a[idx];
    __syncthreads();
    // Compute square in shared memory
    if (idx < N) s_data[threadIdx.x] *= s_data[threadIdx.x];
    __syncthreads();
    // Write result back to global memory
    if (idx < N) a[idx] = s_data[threadIdx.x];
}