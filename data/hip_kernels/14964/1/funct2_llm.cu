#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define inf 9999

__global__ void funct2(int n, int k, float* x, int* qx) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if within bounds
    if (ix < n * n) {
        int j = ix & (n - 1);

        // Use shared memory to reduce global memory access (assuming n is not too large)
        extern __shared__ float shmem[];
        float* sh_x = shmem;
        sh_x[threadIdx.x] = x[ix - j + k];

        __syncthreads();

        float temp2 = sh_x[threadIdx.x] + x[k * n + j];

        if (x[ix] > temp2) {
            x[ix] = temp2;
            qx[ix] = k;
        }
    }
}