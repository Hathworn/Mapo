#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_conv2D_updateDeltas(double* delta, double* biasDelta, const double* upStreamActivation, const double* err, double momentum, size_t kernelCount, size_t kernelRows, size_t kernelCols, size_t outputRows, size_t outputCols, size_t inputChannels, size_t inputRows, size_t inputCols, size_t padding, size_t stride) {
    // Optimize with early exit for out-of-range kernels
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= kernelCount * outputRows * outputCols)
        return;

    // Calculate parallel indices more efficiently
    size_t i = id % outputCols;
    size_t j = (id / outputCols) % outputRows;
    size_t k = id / (outputCols * outputRows);

    // Compute static offsets outside of loops
    size_t outChannelOffset = k * outputRows * outputCols;
    size_t outRowOffset = j * outputCols;
    size_t kk = k * inputChannels * kernelRows * kernelCols;
    size_t index = outChannelOffset + outRowOffset + i;
    int inRowOffset = j * stride - padding;
    int inColOffset = i * stride - padding;

    // Atomic addition to avoid race conditions
    atomicAdd(&biasDelta[k], err[index]);

    for(size_t z = 0; z < inputChannels; z++) {
        size_t kernelChannelOffset = z * kernelRows * kernelCols;
        size_t inChannelOffset = z * inputRows * inputCols;
        for(size_t y = 0; y < kernelRows; y++) {
            int inRow = inRowOffset + y;
            if(inRow >= 0 && inRow < (int)inputRows) { // Check row bounds
                size_t kernelRowOffset = y * kernelCols;
                for(size_t x = 0; x < kernelCols; x++) {
                    int inCol = inColOffset + x;
                    if(inCol >= 0 && inCol < (int)inputCols) { // Check col bounds
                        size_t idx = inChannelOffset + inputCols * inRow + inCol;
                        atomicAdd(&delta[kk + kernelChannelOffset + kernelRowOffset + x], err[index] * upStreamActivation[idx]);
                    }
                }
            }
        }
    }
}