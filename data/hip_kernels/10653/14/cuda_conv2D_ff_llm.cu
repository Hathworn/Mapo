#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_conv2D_ff(double* pA, double* pNet, const double* in, const double* pKernels, const double* pBias, size_t kernelCount, size_t kernelRows, size_t kernelCols, size_t outputRows, size_t outputCols, size_t inputRows, size_t inputCols, size_t inputChannels, size_t padding, size_t stride)
{
    // Calculate global ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    size_t i = id % outputCols;
    id /= outputCols;
    size_t j = id % outputRows;
    id /= outputRows;
    if(id >= kernelCount) return;
    size_t k = id;

    // Pre-calculate channel offsets
    size_t outChannelOffset = k * outputRows * outputCols;
    size_t outRowOffset = j * outputCols;
    int inRowOffset = j * stride - padding;
    size_t kk = k * inputChannels * kernelRows * kernelCols;
    size_t index = outChannelOffset + outRowOffset + i;
    int inColOffset = i * stride - padding;

    // Initialize output with bias
    pNet[index] = pBias[k];

    // Use nested loops for the convolution operation
    for(size_t z = 0; z < inputChannels; z++)
    {
        size_t kernelChannelOffset = z * kernelRows * kernelCols;
        size_t inChannelOffset = z * inputRows * inputCols;
        for(size_t y = 0; y < kernelRows; y++)
        {
            size_t kernelRowOffset = y * kernelCols;
            int inRow = inRowOffset + y;
            for(size_t x = 0; x < kernelCols; x++)
            {
                int inCol = inColOffset + x;
                // Perform bounds check to ensure valid access
                if(inRow >= 0 && inRow < (int)inputRows && inCol >= 0 && inCol < (int)inputCols)
                {
                    size_t idx = inChannelOffset + inputCols * inRow + inCol;
                    // Accumulate result using kernel weights
                    pNet[index] += pKernels[kk + kernelChannelOffset + kernelRowOffset + x] * in[idx];
                }
            }
        }
    }

    // Apply activation function
    pA[index] = tanh(pNet[index]);
}