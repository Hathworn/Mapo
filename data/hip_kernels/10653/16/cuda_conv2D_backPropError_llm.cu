#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_conv2D_backPropError(double* upStreamError, const double* err, const double* pKernels, size_t kernelCount, size_t kernelRows, size_t kernelCols, size_t outputRows, size_t outputCols, size_t inputChannels, size_t inputRows, size_t inputCols, size_t padding, size_t stride)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    size_t i = id % inputChannels;
    id /= inputChannels;
    size_t j = id % outputRows;
    id /= outputRows;
    if(id >= kernelCount)
        return;
    size_t k = id;

    // Precompute offsets for faster access
    size_t outChannelOffset = k * outputRows * outputCols;
    size_t outRowOffset = j * outputCols;
    int inColOffset = i * stride - padding;
    int inRowOffset = j * stride - padding;
    size_t kk = k * inputChannels * kernelRows * kernelCols;
    size_t index = outChannelOffset + outRowOffset + i;

    // Using loop interchange to enhance memory locality
    for(size_t z = 0; z < inputChannels; z++)
    {
        size_t kernelChannelOffset = z * kernelRows * kernelCols;
        size_t inChannelOffset = z * inputRows * inputCols;
        for(size_t y = 0; y < kernelRows; y++)
        {
            int inRow = inRowOffset + y;
            size_t kernelRowOffset = y * kernelCols;
            for(size_t x = 0; x < kernelCols; x++)
            {
                int inCol = inColOffset + x;
                if(inRow >= 0 && inRow < (int)inputRows && inCol >= 0 && inCol < (int)inputCols)
                {
                    size_t idx = inChannelOffset + inputCols * inRow + inCol;
                    // Use atomic addition to avoid race conditions
                    atomicAdd(&upStreamError[idx], pKernels[kk + kernelChannelOffset + kernelRowOffset + x] * err[index]);
                }
            }
        }
    }
}