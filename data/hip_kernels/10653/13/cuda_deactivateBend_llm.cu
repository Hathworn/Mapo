#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_deactivateBend(double* pE, const double* pA, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a register variable to reduce memory access
    if (id < n) {
        double x = pE[id];
        double factor = 0.5 * (x / sqrt(x * x + 1)) + 1;
        pE[id] *= factor;
    }
}