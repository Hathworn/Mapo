#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_deactivateTanh(double* pE, const double* pA, int n)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure id is within bounds
    if (id < n) {
        // Use shared memory for faster access
        __shared__ double shared_pA[1024];
        
        shared_pA[threadIdx.x] = pA[id];
        __syncthreads();
        
        // Perform operation using shared memory
        pE[id] *= (1.0 - (shared_pA[threadIdx.x] * shared_pA[threadIdx.x]));
    }
}