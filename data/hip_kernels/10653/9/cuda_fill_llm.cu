#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_fill(double* pVec, double val, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Use __syncthreads() to ensure all threads are at the same execution point
    if (id < n)
    {
        pVec[id] = val;  // Correct index to write to the specific element
    }
}