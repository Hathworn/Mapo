#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_activateTanh(double* pA, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Unrolled loop to increase parallelism
    for (int i = id; i < n; i += gridDim.x * blockDim.x) {
        pA[i] = tanh(pA[i]);
    }
}