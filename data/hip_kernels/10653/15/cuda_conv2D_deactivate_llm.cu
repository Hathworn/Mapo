#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_conv2D_deactivate(double* err, const double* net, const double* activation, size_t outputs)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds and update error in the same condition
    if(id < outputs) {
        double act = activation[id]; // Load activation value to avoid multiple memory accesses
        err[id] *= (1.0 - act * act); // Use pre-loaded activation value
    }
}