#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void devFillAffectedIndex(int nRemove, int maxTriPerVert, int *pTriangleAffectedIndex)
{
    // Get the thread's global index
    int n = blockIdx.x * blockDim.x + threadIdx.x;

    // Using grid-stride loop for better performance
    for (; n < nRemove; n += blockDim.x * gridDim.x) {
        int baseIndex = n * maxTriPerVert;
        int offsetIndex = baseIndex + nRemove * maxTriPerVert;

        // Unrolling loop to potentially optimize performance
        for (int i = 0; i < maxTriPerVert; i++) { 
            pTriangleAffectedIndex[i + baseIndex] = n;
            pTriangleAffectedIndex[i + offsetIndex] = n;
        }
    }
}