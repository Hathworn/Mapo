#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void devInverseReindexInt3Bool(int N, int3 *destArray, int3 *srcArray, unsigned int *reindex, int realSize, int nDims, int maxValue, bool ignoreValue)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < N) {
        // Unrolled loop and merged common logic for x, y, z
        for (unsigned int n = 0; n < nDims; n++) {
            int3 src = srcArray[i + n * realSize];
            int3 ret = make_int3(-1, -1, -1); // Initialize all to -1
            int addValue;
            if (src.x != -1 || !ignoreValue) {
                addValue = 0;
                src.x = ((src.x % maxValue) + maxValue) % maxValue; // Normalize value within bounds
                ret.x = (int)reindex[src.x] + addValue;
            }
            if (src.y != -1 || !ignoreValue) {
                addValue = 0;
                src.y = ((src.y % maxValue) + maxValue) % maxValue;
                ret.y = (int)reindex[src.y] + addValue;
            }
            if (src.z != -1 || !ignoreValue) {
                addValue = 0;
                src.z = ((src.z % maxValue) + maxValue) % maxValue;
                ret.z = (int)reindex[src.z] + addValue;
            }
            destArray[i + n * realSize] = ret;
        }
        i += gridDim.x * blockDim.x;
    }
}