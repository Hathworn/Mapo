#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void devFindUniqueTriangleAffected(int maxIndex, int *pTriangleAffected, int *pTriangleAffectedIndex, int *pUniqueFlag)
{
    // Using shared memory to cache triangle affected data
    extern __shared__ int sharedMem[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x + 1;

    while (i < maxIndex) {
        // Load data to shared memory
        sharedMem[tid] = pTriangleAffected[i];
        __syncthreads(); // Ensure all threads have loaded their data
        
        if (tid > 0 && sharedMem[tid-1] == sharedMem[tid] && sharedMem[tid] != -1) {
            int j = pTriangleAffectedIndex[i];
            pUniqueFlag[j] = 0;
        }
        __syncthreads(); // Ensure all threads have updated the flag

        i += gridDim.x * blockDim.x;
    }
}