#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize parallelism for better occupancy and performance
__global__ void devInverseReindexInt(int N, int *destArray, int *srcArray, int *reindex, int realSize, int nDims)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll the loop to reduce overhead and improve memory access
    #pragma unroll
    for (unsigned int n = 0; n < nDims; n++) {
        if (i < N) {
            int tmp = srcArray[i + n * realSize];
            destArray[i + n * realSize] = reindex[tmp];
        }
        i += gridDim.x * blockDim.x;
    }
}