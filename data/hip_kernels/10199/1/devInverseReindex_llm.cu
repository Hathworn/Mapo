#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void devInverseReindex(int N, int *destArray, int *srcArray, unsigned int *reindex, int realSize, int nDims, int maxValue, bool ignoreValue)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (unsigned int n = 0; n < nDims; n++) {
        int idx = i;

        while (idx < N) {
            int ret = -1;
            int tmp = srcArray[idx + n * realSize];
            
            if (tmp != -1 || !ignoreValue) {
                
                // Normalize tmp to be within bounds [0, maxValue)
                int addValue = (tmp < 0) ? -((-tmp / maxValue) + 1) * maxValue : (tmp / maxValue) * maxValue;
                tmp -= addValue;
                
                // Adjust tmp if it is negative after normalization
                while (tmp < 0) {
                    tmp += maxValue;
                    addValue -= maxValue;
                }

                ret = (int) reindex[tmp] + addValue;
            }
            destArray[idx + n * realSize] = ret;

            idx += stride;
        }
    }
}