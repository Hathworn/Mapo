#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "devInverseReindexInt3_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    int N = XSIZE*YSIZE;
int3 *destArray = NULL;
hipMalloc(&destArray, XSIZE*YSIZE);
int3 *srcArray = NULL;
hipMalloc(&srcArray, XSIZE*YSIZE);
int *reindex = NULL;
hipMalloc(&reindex, XSIZE*YSIZE);
int realSize = XSIZE*YSIZE;
int nDims = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
devInverseReindexInt3<<<gridBlock, threadBlock>>>(N,destArray,srcArray,reindex,realSize,nDims);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
devInverseReindexInt3<<<gridBlock, threadBlock>>>(N,destArray,srcArray,reindex,realSize,nDims);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
devInverseReindexInt3<<<gridBlock, threadBlock>>>(N,destArray,srcArray,reindex,realSize,nDims);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}