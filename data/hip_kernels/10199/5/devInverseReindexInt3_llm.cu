#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void devInverseReindexInt3(int N, int3 *destArray, int3 *srcArray, int *reindex, int realSize, int nDims)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index

    for (unsigned int n = 0; n < nDims; n++) {
        int idx = i + n * realSize; // Calculate index for dimension n

        while (i < N) {
            // Efficient memory access using local variables
            int3 temp = srcArray[idx];

            // Avoid redundant accesses by using local variables
            destArray[idx].x = reindex[temp.x];
            destArray[idx].y = reindex[temp.y];
            destArray[idx].z = reindex[temp.z];

            i += gridDim.x * blockDim.x; // Update index for next iteration
            idx += gridDim.x * blockDim.x; // Update idx for next iteration
        }
    }
}