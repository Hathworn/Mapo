#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void devInverseReindexInt2Bool(int N, int2 *destArray, int2 *srcArray, unsigned int *reindex, int realSize, int nDims, int maxValue, bool ignoreValue)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int totalThreads = gridDim.x * blockDim.x;

    // Loop over dimensions with stride to improve parallel work distribution
    for (unsigned int n = 0; n < nDims; n++) {
        while (i < N) {
            int ret = -1;
            int tmp = srcArray[i + n*realSize].x;
            if (tmp != -1 || ignoreValue == false) {
                ret = tmp % maxValue;  // Perform modulo operation directly
                if (ret < 0) ret += maxValue;  // Adjust negative numbers
                ret = (int) reindex[ret] + (tmp / maxValue) * maxValue;  // Calculate final index with offset
            }
            destArray[i + n*realSize].x = ret;

            ret = -1;
            tmp = srcArray[i + n*realSize].y;
            if (tmp != -1 || ignoreValue == false) {
                ret = tmp % maxValue;  // Perform modulo operation directly
                if (ret < 0) ret += maxValue;  // Adjust negative numbers
                ret = (int) reindex[ret] + (tmp / maxValue) * maxValue;  // Calculate final index with offset
            }
            destArray[i + n*realSize].y = ret;

            i += totalThreads;  // Move to the next index in the stride
        }
    }
}