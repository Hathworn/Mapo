#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "devFillAffectedTriangles_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    int nFlip = 1;
int *pTaff = NULL;
hipMalloc(&pTaff, XSIZE*YSIZE);
int *pTaffEdge = NULL;
hipMalloc(&pTaffEdge, XSIZE*YSIZE);
int *pEnd = NULL;
hipMalloc(&pEnd, XSIZE*YSIZE);
int2 *pEt = NULL;
hipMalloc(&pEt, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
devFillAffectedTriangles<<<gridBlock, threadBlock>>>(nFlip,pTaff,pTaffEdge,pEnd,pEt);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
devFillAffectedTriangles<<<gridBlock, threadBlock>>>(nFlip,pTaff,pTaffEdge,pEnd,pEt);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
devFillAffectedTriangles<<<gridBlock, threadBlock>>>(nFlip,pTaff,pTaffEdge,pEnd,pEt);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}