#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void devFillAffectedTriangles(int nFlip, int *pTaff, int *pTaffEdge, int *pEnd, int2 *pEt)
{
    // Calculate global thread index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Utilize grid stride loop for enhanced parallelism
    for (; i < nFlip; i += gridDim.x * blockDim.x) {
        int e = pEnd[i];

        // Concurrent access to memory with coalesced approach
        pTaffEdge[i] = i;
        pTaffEdge[i + nFlip] = i;

        pTaff[i] = pEt[e].x;
        pTaff[i + nFlip] = pEt[e].y;
    }
}