#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// ERROR system
#define EXIT_OK (0)
#define ERROR_HOST_MEM (1)
#define ERROR_DEVICE_MEM (2)
#define ERROR_DEVICE (3)
#define ERROR_INIT (4)
#define ERROR_EMPTY (5)
#define ERROR_SETDEVICE (6)
#define EXIT_DONE (255)

// kernel to calculate the exp

__global__ void kmeans_exp_kernel(float* DIST, float pw)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplified register usage
    if (idx < gridDim.x * blockDim.x) { // Boundary check to avoid out-of-bounds access
        float arg = DIST[idx] * pw;
        if (arg < -70) arg = -70; // Avoid passing an overly large negative value to exp
        DIST[idx] = expf(arg); // Use expf for better performance with float
    }
}