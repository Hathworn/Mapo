#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rotate_180( float* data,int nx, int nxy, int offset, unsigned int size) {

    const uint x = threadIdx.x;
    const uint y = blockIdx.x;

    // Use dynamic shared memory to avoid excess usage
    extern __shared__ float shared_data[];

    float* shared_lower_data = shared_data;
    float* shared_upper_data = shared_data + blockDim.x;

    // Load data into shared memory with coalesced access
    shared_lower_data[x] = data[x + y * MAX_THREADS + offset];
    shared_upper_data[x] = data[nxy + x + (-y - 1) * MAX_THREADS - offset];
    __syncthreads();

    // Swap using shared memory
    if (x < size) {
        float tmp = shared_lower_data[x];
        shared_lower_data[x] = shared_upper_data[MAX_THREADS - x - 1];
        shared_upper_data[MAX_THREADS - x - 1] = tmp;
    }
    __syncthreads();

    // Write back to global memory with coalesced access
    if (x < size) {
        data[nxy - x - 1 + (-y) * MAX_THREADS - offset] = shared_upper_data[MAX_THREADS - x - 1];
        data[x + y * MAX_THREADS + offset] = shared_lower_data[x];
    }
}