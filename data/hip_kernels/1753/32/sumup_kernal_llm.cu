#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumup_kernal(const float * data, float * device_stats, const int size, const int dim2size, const int num_threads, const int offset)
{
    float sum = 0.0f;

    const uint x = threadIdx.x;
    const uint y = blockIdx.x;

    int idx = x + y * num_threads + offset;

    // Unroll loop for optimization
    for (int i = 0; i < size; i += 4) { 
        int index1 = i * dim2size + idx % dim2size;
        sum += data[index1];

        // Preventing out-of-bounds access with conditional loading
        if (i+1 < size) {
            int index2 = (i+1) * dim2size + idx % dim2size;
            sum += data[index2];
        }

        if (i+2 < size) {
            int index3 = (i+2) * dim2size + idx % dim2size;
            sum += data[index3];
        }

        if (i+3 < size) {
            int index4 = (i+3) * dim2size + idx % dim2size;
            sum += data[index4];
        }
    }

    // Store the result in global memory
    device_stats[idx] = sum / size;
}