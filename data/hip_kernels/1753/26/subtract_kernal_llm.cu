#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subtract_kernal(float* data, float f, const int totaltc)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within bounds for processing
    if(idx < totaltc){
        data[idx] -= f; // Perform subtraction operation
    }
}