#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void swap_middle_row(float* data, const int num_threads, const int nx, const int ny, const int xodd, const int yodd, const int offset) {
    // Use shared memory for reduction of global memory access
    __shared__ float shared_data[1024]; // assuming block size is 1024

    const uint x = threadIdx.x;
    const uint y = blockIdx.x;

    const uint c = x + y * num_threads + offset;
    int r = ny / 2;
    int idx1 = r * nx + c;
    int idx2 = r * nx + c + nx / 2 + xodd;

    // Load global data to shared memory
    shared_data[x] = data[idx1];
    __syncthreads();

    // Perform swap using shared memory
    float tmp = shared_data[x];
    data[idx1] = data[idx2];
    data[idx2] = tmp;
}