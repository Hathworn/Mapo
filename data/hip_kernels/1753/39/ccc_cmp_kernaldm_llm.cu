#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ccc_cmp_kernaldm(const float* __restrict__ data1, const float* __restrict__ data2, const float* __restrict__ dm, float* __restrict__ device_soln, const int size, const int num_calcs, const int num_threads, const int offset)
{
    // Use registers for accumulated values
    float avg1 = 0.0f;
    float avg2 = 0.0f;
    float var1 = 0.0f;
    float var2 = 0.0f;
    float ccc = 0.0f;
    float nnn = 0.0f;

    const uint x = threadIdx.x;
    const uint y = blockIdx.x;
    int idx = x + y * num_threads + offset;

    // Unroll loop for better performance
    #pragma unroll
    for(int i = 0; i < size; i++) {
        int index = i * size + idx % size + ((idx / size) * size * size); // For coalescing

        if(dm[index] > 0.5) {
            const float val1 = data1[index];
            const float val2 = data2[index];

            avg1 += val1;
            avg2 += val2;
            var1 += val1 * val1;
            var2 += val2 * val2;
            ccc += val1 * val2;
            nnn += 1.0f;
        }
    }

    // Avoid division by zero
    if (nnn > 0.0f) {
        device_soln[idx] = avg1 / nnn;
        device_soln[idx + num_calcs] = avg2 / nnn;
        device_soln[idx + 2 * num_calcs] = var1 / nnn;
        device_soln[idx + 3 * num_calcs] = var2 / nnn;
        device_soln[idx + 4 * num_calcs] = ccc / nnn;
    }
}