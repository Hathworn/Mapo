#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Constants are pre-calculated for efficient computation
__global__ void mul_ctf(float *image, int nx, int ny, float defocus, float cs, float voltage, float apix, float bfactor, float ampcont) {

    // Block index
    int bx = blockIdx.x;

    // Thread index
    int tx = threadIdx.x;

    // Compute coordinates
    float x = float(bx);
    float y = (tx >= ny >> 1) ? float(tx - ny) : float(tx);
    int index = bx * 2 + tx * (nx + 2);

    // Pre-calculate constants for CTF computation
    float lambda = 12.398f / sqrt(voltage * (1022.f + voltage));
    float cst = cs * 1.0e7f;
    float wgh = ampcont / 100.0f;
    float phase = atan(wgh / sqrt(1.0f - wgh * wgh));
    float ak = sqrt(x * x + y * y) / (nx * apix);
    float ak2 = ak * ak;

    // Compute CTF
    float g1 = defocus * 1.0e4f * lambda * ak2;
    float g2 = cst * lambda * lambda * lambda * ak2 * ak2 * 0.5f;
    float ctfv = sin(PI * (g1 - g2) + phase);
    
    if (bfactor != 0.0f) {
        ctfv *= exp(-bfactor * ak2 * 0.25f);
    }

    // Apply CTF to the complex image
    image[index] *= ctfv;
    image[index + 1] *= ctfv;
}