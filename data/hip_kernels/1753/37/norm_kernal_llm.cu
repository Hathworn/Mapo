#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_THREADS 256

__global__ void norm_kernal(float * data, float mean, float var, int totaltc)
{
    // Use shared memory to read input data if possible
    __shared__ float sharedData[MAX_THREADS];
    const uint index = threadIdx.x + (blockIdx.x + gridDim.x*blockIdx.y)*MAX_THREADS;

    // Check bounds to prevent overwriting out-of-bound memory
    if(index < totaltc)
    {
        // Load data into shared memory
        sharedData[threadIdx.x] = data[index];
        __syncthreads(); // Ensure all threads have completed the data load
        
        // Perform normalization using shared memory
        sharedData[threadIdx.x] = (sharedData[threadIdx.x] - mean) / var;
        
        // Write result back to global memory
        data[index] = sharedData[threadIdx.x];
    }
}