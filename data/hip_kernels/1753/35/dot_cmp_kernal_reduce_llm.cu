#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_cmp_kernal_reduce(float *g_idata1, float *g_idata2, float *g_odata)
{
    extern __shared__ float sdata[];

    // Load data from global to shared mem with optimized coalesced access
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * (blockDim.x * 2) + tid;
    
    float sum = 0.0f;
    if (idx < blockDim.x * gridDim.x) {
        // Combine two load operations and avoid out of bounds access
        sum = g_idata1[idx] * g_idata2[idx] + g_idata1[idx + blockDim.x] * g_idata2[idx + blockDim.x];
    }
    sdata[tid] = sum;
    __syncthreads();

    // Unroll the last warp for better performance
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Warp unrolling for last 32 threads
    if (tid < 32) {
        if (blockDim.x >= 64) sdata[tid] += sdata[tid + 32];
        if (blockDim.x >= 32) sdata[tid] += sdata[tid + 16];
        if (blockDim.x >= 16) sdata[tid] += sdata[tid + 8];
        if (blockDim.x >= 8) sdata[tid] += sdata[tid + 4];
        if (blockDim.x >= 4) sdata[tid] += sdata[tid + 2];
        if (blockDim.x >= 2) sdata[tid] += sdata[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}