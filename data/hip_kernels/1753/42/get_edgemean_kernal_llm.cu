#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void get_edgemean_kernal(const float* data, float* edgemean, const int nx, const int ny, const int nz)
{
    // Precompute frequent calculations
    size_t nxy = nx * ny;
    float edge_sum = 0.0f;

    if (nz == 1) {
        // Use a single loop for contiguous memory access
        for (int i = 0, j = (ny - 1) * nx; i < nx; ++i, ++j) {
            edge_sum += data[i] + data[j];
        }
        for (size_t i = 0, j = nx - 1; i < nxy; i += nx, j += nx) {
            edge_sum += data[i] + data[j];
        }
        *edgemean = edge_sum / (nx * 2 + ny * 2); // Final calculation of mean
        return;
    }

    int di = 0;
    size_t offset_nz = (nz - 1) * nxy;

    // Handle special cube condition
    if (nx == ny && nx == nz * 2 - 1) {
        for (size_t j = offset_nz; j < nxy * nz; ++j, ++di) {
            edge_sum += data[j];
        }
    } else {
        for (size_t i = 0, j = offset_nz; i < nxy; ++i, ++j, ++di) {
            edge_sum += data[i] + data[j];
        }
    }

    int nxy2 = nx * (ny - 1);
    for (int k = 1; k < nz - 1; ++k) {
        size_t k2 = k * nxy;
        size_t k3 = k2 + nxy2;
        for (int i = 0; i < nx; ++i, ++di) {
            edge_sum += data[i + k2] + data[i + k3];
        }
    }
    for (int k = 1; k < nz - 1; ++k) {
        size_t k2 = k * nxy;
        size_t k3 = nx - 1 + k2;
        for (int i = 1; i < ny - 1; ++i, ++di) {
            edge_sum += data[i * nx + k2] + data[i * nx + k3];
        }
    }

    *edgemean = edge_sum / (di * 2); // Final calculation of mean
}