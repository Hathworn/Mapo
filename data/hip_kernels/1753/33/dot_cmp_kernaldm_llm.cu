#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot_cmp_kernaldm(const float* __restrict__ data1, const float* __restrict__ data2, const float* __restrict__ dm, float* __restrict__ device_soln, const int size, const int num_threads, const int offset)
{
    float dot = 0.0f;
    float nnn = 0.0f;

    int idx = threadIdx.x + blockIdx.x * num_threads + offset;
    int stride = blockDim.x * gridDim.x; // Use the grid stride loop

    // Optimize access pattern to utilize shared memory or cache more efficiently
    for (int i = idx; i < size * size; i += stride) {
        int index = i % size + ((i / size) * size * size);
        if(dm[index] > 0.5){
            dot += data1[index] * data2[index];
            nnn += 1.0f;
        }
    }

    device_soln[idx] = dot / (nnn > 0.0f ? nnn : 1.0f); // Handle division by zero safely
}