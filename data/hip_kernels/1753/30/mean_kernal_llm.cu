#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernal(const float * data, float * device_stats, const int size, const int num_calcs, const int num_threads, const int offset)
{
    // Precompute repeated operations outside the loop for efficiency
    const uint x = threadIdx.x;
    const uint y = blockIdx.x;
    int base_idx = x + y * num_threads + offset;
    int base_stride = base_idx / size * size * size;
    int base_mod_size = base_idx % size;

    float mean = 0.0f;

    // Loop unrolling and memory coalescing
    for (int i = 0; i < size; i += 4) {
        int index = i * size + base_mod_size + base_stride;
        mean += data[index];
        if (i + 1 < size) mean += data[index + size];
        if (i + 2 < size) mean += data[index + 2 * size];
        if (i + 3 < size) mean += data[index + 3 * size];
    }

    device_stats[base_idx] = mean / size;
}