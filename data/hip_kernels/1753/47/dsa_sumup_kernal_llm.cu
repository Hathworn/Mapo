#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dsa_sumup_kernal(float* results, const int inx, const int iny)
{
    extern __shared__ float shared[]; // Use shared memory for partial sums
    int tidx = threadIdx.x;
    int bd = blockDim.x;
    int size = iny * (inx / 2 + 1);

    float dot = 0.0f;
    float vweight = 0.0f;
    float power = 0.0f;
    float power2 = 0.0f;

    for (int i = tidx; i < (inx / 2 + 1); i += bd) { // Loop unrolling by using thread-specific stride
        int idx = i;
        dot += results[idx];
        vweight += results[size + idx];
        power += results[2 * size + idx];
        power2 += results[3 * size + idx];
    }

    // Reduce partial sums within block
    shared[tidx] = dot;
    shared[bd + tidx] = vweight;
    shared[2 * bd + tidx] = power;
    shared[3 * bd + tidx] = power2;
    __syncthreads();

    // Sequential addressing reduction
    for (int offset = bd / 2; offset > 0; offset >>= 1) {
        if (tidx < offset) {
            shared[tidx] += shared[tidx + offset];
            shared[bd + tidx] += shared[bd + tidx + offset];
            shared[2 * bd + tidx] += shared[2 * bd + tidx + offset];
            shared[3 * bd + tidx] += shared[3 * bd + tidx + offset];
        }
        __syncthreads();
    }

    // Output the reduced result to global memory
    if (tidx == 0) {
        results[0] = shared[0];
        results[size] = shared[bd];
        results[2 * size] = shared[2 * bd];
        results[3 * size] = shared[3 * bd];
    }
}