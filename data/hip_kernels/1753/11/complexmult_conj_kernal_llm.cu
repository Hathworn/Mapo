#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void complexmult_conj_kernal(float *afft, const float *bfft, int totaltc)
{
    // Calculate index ensuring coalesced memory access
    const uint idx = threadIdx.x + (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x;
    const uint ridx = 2 * idx; // Separate real and imaginary parts

    if (ridx < totaltc) {
        // Load complex numbers using float2 for better performance
        float2 a = *((float2*)&afft[ridx]);
        float2 b = *((float2*)&bfft[ridx]);

        // Perform complex multiplication with conjugated second operand
        afft[ridx] = a.x * b.x + a.y * b.y;    // Real part
        afft[ridx + 1] = a.y * b.x - a.x * b.y; // Imaginary part
    }
}