#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mcfauto_kernal(const float* data1, float* data2, const int totaltc)
{
    // Calculate the global thread index
    int idx = 2 * (threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * gridDim.x * blockDim.x);

    // Ensure idx is within bounds
    if (idx < totaltc) {
        // Perform calculations and optimize by using local register for intermediate storage
        float data1_idx = data1[idx];
        float data1_idx1 = data1[idx + 1];
        float data2_idx = data2[idx];
        float data2_idx1 = data2[idx + 1];

        // Compute and assign result
        data2[idx] = sqrt(data1_idx * data2_idx + data1_idx1 * data2_idx1);

        // Set next element to zero
        data2[idx + 1] = 0;
    }
}