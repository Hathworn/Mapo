#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernal_reduce(float *g_idata, float *g_odata)
{
    extern __shared__ float sdata[];
    // Calculate thread and element indices
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;

    // Load elements into shared memory, check array boundary
    sdata[tid] = (i < blockDim.x * gridDim.x) ? g_idata[i] : 0;
    sdata[tid] += ((i+blockDim.x) < blockDim.x * gridDim.x) ? g_idata[i+blockDim.x] : 0;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}