#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_kernel(float* data, const float scale, const int realtc)
{
    // Calculate global thread index for the current thread
    const uint index = blockIdx.x * blockDim.x + threadIdx.x + (gridDim.x * blockDim.x * blockIdx.y);

    // Check within bounds and perform scaling operation
    if (index < realtc)
    {
        data[index] *= scale;
    }
}