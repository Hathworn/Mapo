#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void exp_kernel(float* DIST, float pw)
{
    register int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= gridDim.x * blockDim.x) return; // Boundary check
    register float arg = DIST[idx] * pw;
    arg = fmaxf(arg, -70.0f); // Use fmaxf for branchless operation
    DIST[idx] = expf(arg); // Use expf for better performance
}