#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void meansquare_kernal(const float * data, float * device_stats, const int size, const int num_calcs, const int num_threads, const int offset)
{
    float meansq = 0.0f;
    const uint x = threadIdx.x;
    const uint y = blockIdx.x;
    int idx = x + y * num_threads + offset;

    // Use shared memory for partial reduction to improve memory access efficiency
    extern __shared__ float shared_data[];
    shared_data[threadIdx.x] = 0.0f;

    for(int i = 0; i < size; i++){
        int index = i * size + idx % size + ((idx / size) * size * size);
        shared_data[threadIdx.x] += data[index] * data[index];
    }

    __syncthreads(); // Ensure all threads have written to shared memory

    // Perform block-wise reduction to compute the mean square
    if (threadIdx.x == 0) {
        for (int i = 1; i < blockDim.x; i++) {
            shared_data[0] += shared_data[i];
        }
        device_stats[idx] = shared_data[0] / size;
    }
}