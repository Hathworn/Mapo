#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void column_sum(const float* data, float* sum, int nx, int ny, int num_threads, int offset) {

    float s = 0.0f;
    // Utilize grid-stride loop for better occupancy and coalescing
    for (uint idx = threadIdx.x + blockIdx.x * num_threads + offset; idx < nx; idx += gridDim.x * num_threads) {
        for (int i = 0; i < ny; i++) {
            s += data[idx + i * nx];
        }
        sum[idx] = s;
    }
}