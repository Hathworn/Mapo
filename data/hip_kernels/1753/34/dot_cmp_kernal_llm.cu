#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_cmp_kernal(const float* data1, const float* data2, float* device_soln, const int size, const int num_threads, const int offset)
{
    extern __shared__ float shared_data[]; // Use shared memory for partial sums
    float dot = 0.0f;

    int idx = threadIdx.x + blockIdx.x * num_threads + offset;

    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        int index = i * size + idx % size + ((idx / size) * size * size); // Improved memory access pattern for better coalescing
        dot += data1[index] * data2[index];
    }

    shared_data[threadIdx.x] = dot; // Store result in shared memory
    __syncthreads(); // Synchronize threads within block

    // Reduce partial sums within the block
    if (threadIdx.x == 0) {
        float block_dot = 0.0f;
        for (int i = 0; i < blockDim.x; i++) {
            block_dot += shared_data[i];
        }
        device_soln[blockIdx.x] = block_dot / size;
    }
}