#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void swap_top_left_bot_right(float* data, const int num_threads, const int nx, const int ny, const int xodd, const int yodd, const int offset) {
    // Efficiently calculate global index
    const uint gpu_idx = threadIdx.x + blockIdx.x * num_threads + offset;
    
    if (gpu_idx < (nx/2) * (ny/2) + yodd) {  // Bounds check for valid indices
        const uint c = gpu_idx % (nx/2);
        const uint r = gpu_idx / (nx/2) + ny / 2 + yodd;

        const uint idx1 = r * nx + c;
        const uint idx2 = (r - ny / 2 - yodd) * nx + c + nx / 2 + xodd;

        // Swap values using fewer memory accesses
        float tmp = data[idx1];
        data[idx1] = data[idx2];
        data[idx2] = tmp;
    }
}