#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void swap_middle_column(float* data, const int num_threads, const int nx, const int ny, const int xodd, const int yodd, const int offset) {
    const uint x = threadIdx.x;
    const uint y = blockIdx.x;

    const uint r = x + y * num_threads + offset;
    const int c = nx / 2;
    if (r < ny / 2) { // Ensure r is within bounds
        int idx1 = r * nx + c;
        int idx2 = (r + ny / 2 + yodd) * nx + c;
        float tmp = data[idx1];
        data[idx1] = data[idx2];
        data[idx2] = tmp;
    }
}