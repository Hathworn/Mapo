#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_img(float *image_padded, float *ave1, float *ave2, int nx, int ny, int nima) {

    // Optimized index calculations and memory coalescing
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    
    int index = tx + bx * nx;
    int stride = (nx * 2 + 2) * ny * 2;
    int index2 = tx + (nx >> 1) + (bx + (ny >> 1)) * (nx * 2 + 2);
    
    float sum1 = 0.0f;
    float sum2 = 0.0f;

    // Unroll loop for better performance
    for (int i = 0; i < nima; i += 2) {
        sum1 += image_padded[index2 + i * stride];
    }
    for (int i = 1; i < nima; i += 2) {
        sum2 += image_padded[index2 + i * stride];
    }

    // Store results
    ave1[index] = sum1;
    ave2[index] = sum2;
}