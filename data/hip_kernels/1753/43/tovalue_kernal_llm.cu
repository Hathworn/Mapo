#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tovalue_kernal(float* data, const float value, const int totaltc)
{
    // Calculate the global index using block and grid dimensions
    const uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if idx is within bounds and perform assignment
    if (idx < totaltc) {
        data[idx] = value;
    }
}