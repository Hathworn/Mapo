#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stats_kernal(const float *data, float *device_soln, const int size, const int num_calcs, const int num_threads, const int offset)
{
    // Declare shared memory for block-level reduction
    extern __shared__ float shared_data[];

    float sum = 0.0f;
    float sum_sq = 0.0f;

    int idx = threadIdx.x + blockIdx.x * num_threads + offset;

    for (int i = 0; i < size; i++)
    {
        int index = i * size + idx % size + ((idx / size) * size * size); 
        float datum = data[index];
        sum += datum;
        sum_sq += datum * datum;
    }

    shared_data[threadIdx.x] = sum;
    shared_data[threadIdx.x + blockDim.x] = sum_sq;

    __syncthreads();

    // Reduction in shared memory
    if (threadIdx.x == 0)
    {
        for (int i = 1; i < blockDim.x; i++)
        {
            sum += shared_data[i];
            sum_sq += shared_data[i + blockDim.x];
        }
        device_soln[idx] = sum;
        device_soln[idx + num_calcs] = sum_sq;
    }
}