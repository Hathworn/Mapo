#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void middle_to_right(float* data, const int nx, const int ny) 
{
    // Cache thread index and stride for global memory access
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int stride = blockDim.y * gridDim.y;

    // Process data row by row
    for (; r < ny; r += stride) {
        float last_val = data[r * nx + nx / 2];
        
        // Use a reverse loop over columns starting from nx/2
        for (int c = nx - 1; c >= nx / 2; --c) {
            int idx = r * nx + c;
            float tmp = data[idx];
            data[idx] = last_val;
            last_val = tmp;
        }
    }
}