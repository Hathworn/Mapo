#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult_complex_eff_kernal(float* data, const float* src_data, const int nx, const int nxy, const int size)
{
    int idx = blockIdx.z*blockDim.z*nxy + blockIdx.y*blockDim.y*nx + blockIdx.x*blockDim.x 
              + threadIdx.z*nxy + threadIdx.y*nx + threadIdx.x;
    
    // Ensure idx is within valid range
    if (idx < size / 2) {
        data[idx] *= src_data[idx];
        data[size-idx-1] *= src_data[size-idx-1];
    }
}