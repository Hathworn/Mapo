#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ccc_cmp_kernal(const float* data1, const float* data2, float* device_soln, const int size, const int num_calcs, const int num_threads, const int offset)
{
    // Use shared memory to reduce global memory accesses
    extern __shared__ float shared_mem[];

    float* shared_data1 = shared_mem;
    float* shared_data2 = shared_mem + size;

    float avg1 = 0.0f;
    float avg2 = 0.0f;
    float var1 = 0.0f;
    float var2 = 0.0f;
    float ccc = 0.0f;

    const uint x = threadIdx.x;
    const uint y = blockIdx.x;
    int idx = x + y * num_threads + offset;

    // Load data into shared memory
    for(int i = x; i < size; i += num_threads) {
        int index = i * size + idx % size + ((idx / size) * size * size);
        shared_data1[i] = data1[index];
        shared_data2[i] = data2[index];
    }
    __syncthreads();

    // Perform calculations using shared memory
    for(int i = 0; i < size; i++){
        avg1 += shared_data1[i];
        avg2 += shared_data2[i];
        var1 += shared_data1[i] * shared_data1[i];
        var2 += shared_data2[i] * shared_data2[i];
        ccc += shared_data1[i] * shared_data2[i];
    }

    device_soln[idx] = avg1 / size;
    device_soln[idx + num_calcs] = avg2 / size;
    device_soln[idx + 2 * num_calcs] = var1 / size;
    device_soln[idx + 3 * num_calcs] = var2 / size;
    device_soln[idx + 4 * num_calcs] = ccc / size;
}