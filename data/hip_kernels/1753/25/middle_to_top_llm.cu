#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void middle_to_top(float* data, const int nx, const int ny)
{
    // Calculate column position for each thread
    int c = threadIdx.x + blockIdx.x * blockDim.x;
    if (c < nx) {
        // Get the value in the top row
        float last_val = data[ny/2 * nx + c];
        // Iterate over rows from middle to top
        for (int r = ny - 1; r >= ny / 2; --r) {
            int idx = r * nx + c;
            float tmp = data[idx];
            data[idx] = last_val;
            last_val = tmp;
        }
    }
}