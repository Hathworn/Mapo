#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void swap_bot_left_top_right(float* data, const int num_threads, const int nx, const int ny, const int xodd, const int yodd, const int offset) {
    const uint x = threadIdx.x;
    const uint y = blockIdx.x;
    
    // Calculate global index directly using faster operations
    const uint gpu_idx = x + y * num_threads + offset;
    const uint row_size_half = nx / 2;
    
    // Compute column and row indices more efficiently
    const uint c = gpu_idx % row_size_half;
    const uint r = gpu_idx / row_size_half;
    
    // Pre-calculate reused values
    const uint r_offset = r * nx;
    const uint idx1 = r_offset + c;
    const uint idx2 = (r + ny / 2 + yodd) * nx + c + row_size_half + xodd;
    
    // Perform the swap operation
    float tmp = data[idx1];
    data[idx1] = data[idx2];
    data[idx2] = tmp;
}