#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void complexmult_kernal(float *afft, const float *bfft, int totaltc)
{
    const uint tid = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * gridDim.x * blockDim.x;
    const uint ridx = 2 * tid;

    if (ridx < totaltc) {
        const uint iidx = ridx + 1;

        // Use float2 for coalesced memory access
        float2 a = *reinterpret_cast<float2*>(&afft[ridx]);
        float2 b = *reinterpret_cast<const float2*>(&bfft[ridx]);

        // Complex multiplication using float2
        afft[ridx] = a.x * b.x - a.y * b.y;   // real part
        afft[iidx] = a.y * b.x + a.x * b.y;   // imaginary part
    }
}