#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel3(int* D, int* q, int b) {
    // Calculate indices outside of conditional for better readability
    int blockOffsetX = blockIdx.x * blockDim.x + threadIdx.x;
    int blockOffsetY = blockIdx.y * blockDim.y + threadIdx.y;

    // Use ternary operators for concise index calculation
    int i = blockIdx.x >= b ? blockOffsetX + blockDim.x : blockOffsetX;
    int j = blockIdx.y >= b ? blockOffsetY + blockDim.y : blockOffsetY;

    // Store N and THR_PER_BL in shared memory for faster access
    __shared__ int shared_N;
    __shared__ int shared_THR_PER_BL;
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        shared_N = N;
        shared_THR_PER_BL = THR_PER_BL;
    }
    __syncthreads();

    float d, f, e;
    // Loop condition simplified
    for (int k = b * shared_THR_PER_BL; k < (b + 1) * shared_THR_PER_BL; k++) {
        d = D[i * shared_N + j];
        f = D[i * shared_N + k];
        e = D[k * shared_N + j];

        __syncthreads(); // Ensure all threads have loaded their elements

        // Use simplified condition
        if (d > f + e) {
            D[i * shared_N + j] = f + e;
            q[i * shared_N + j] = k;
        }
    }
}