#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(int* D, int* q, int k) {
    // Use shared memory for common element accesses
    extern __shared__ int shared_mem[];
    int* sD = shared_mem;
    
    // Find index of i row and j column of the distance array
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Load current matrix row and column into shared memory
    int index_ik = i * N + k;
    int index_kj = k * N + j;
    int Dij = i * N + j;

    __syncthreads();
    
    if (D[Dij] > D[index_ik] + D[index_kj]) {
        D[Dij] = D[index_ik] + D[index_kj];
        q[Dij] = k;
    }
}