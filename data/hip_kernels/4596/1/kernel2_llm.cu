#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// Copyright 2019, Dimitra S. Kaitalidou, All rights reserved

#define N 256
#define THR_PER_BL 8
#define BL_PER_GR 32

__global__ void kernel2(int* D, int* Q){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Prevent out-of-bounds access
    if (i < N) {
        D[i] = Q[i];
    }
}