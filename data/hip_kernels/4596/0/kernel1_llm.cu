#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// Copyright 2019, Dimitra S. Kaitalidou, All rights reserved

#define N 256
#define THR_PER_BL 8
#define BL_PER_GR 32

__global__ void kernel1(int* D, int* Q, int k) {
    // Calculate the thread's index within the grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Direct calculation of block index based on i and k
    int block = i / (2 * k);

    // Calculate corresponding index j
    int j = (i % 2 == 0) ? 
        (2 * block * k + i / 2 - k * (i / (2 * k))) : 
        ((2 * block + 1) * k + i / 2 - k * (i / (2 * k)));
    
    // Assign the values to the output array
    Q[j] = D[i];
}