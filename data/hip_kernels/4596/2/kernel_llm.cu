#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(int* D, int* Q, int bits) {
    // Compute the index using block and thread information
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform bit reversal using a more efficient approach
    int shifted_left = 0;
    for (int a = 0; a < bits; a++) {
        shifted_left |= (i & 1) << (bits - 1 - a);
        i >>= 1;
    }
    
    // Assign the values to the bit-reversed positions
    Q[shifted_left] = D[blockIdx.x * blockDim.x + threadIdx.x];
}