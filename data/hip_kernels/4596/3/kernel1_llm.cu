#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel1(int* D, int* q, int b) {
    int i = threadIdx.x + b * THR_PER_BL;
    int j = threadIdx.y + b * THR_PER_BL;

    // Use shared memory to reduce global memory accesses
    __shared__ float sharedD[THR_PER_BL][THR_PER_BL];

    float d, f, e;
    d = D[i * N + j];
    
    for (int k = b * THR_PER_BL; k < (b + 1) * THR_PER_BL; k++) {
        // Load data into shared memory
        sharedD[threadIdx.x][threadIdx.y] = D[i * N + k];
        __syncthreads();

        f = sharedD[threadIdx.x][threadIdx.y];
        e = D[k * N + j];

        // Synchronize to ensure all threads have loaded their data
        __syncthreads();

        if (d > f + e) {
            // Perform update in register before writing to global memory
            d = f + e;
            D[i * N + j] = d;
            q[i * N + j] = k;
        }
    }
}