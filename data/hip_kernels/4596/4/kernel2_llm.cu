#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel2(int* D, int* q, int b) {
    // Pre-calculate index using block and thread dimensions for efficiency
    int i = (blockIdx.y == 0) ? ((blockIdx.x >= b) ? (blockIdx.x + 1) * blockDim.x + threadIdx.x : blockIdx.x * blockDim.x + threadIdx.x) 
                              : b * blockDim.y + threadIdx.y;
    int j = (blockIdx.y == 0) ? b * blockDim.y + threadIdx.y 
                              : ((blockIdx.x >= b) ? (blockIdx.x + 1) * blockDim.x + threadIdx.x : blockIdx.x * blockDim.x + threadIdx.x);

    float d, f, e;
    // Avoid global memory access for unchanged data by prefetching inside loop
    for(int k = b * THR_PER_BL; k < (b + 1) * THR_PER_BL; k++) {
        d = D[i * N + j];
        f = D[i * N + k];
        e = D[k * N + j];

        __syncthreads(); // Ensure all threads have fetched these values

        if(d > f + e) {
            D[i * N + j] = f + e; // Update only if condition is met
            q[i * N + j] = k;     // Update path with current k
        }
    }
}