#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Safeguard to prevent division by zero
    if (N != 0) {
        *accuracy /= N;
    }
}