#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups() {
    // No operation needed in this function – function is a sync point
    __syncthreads();  // Ensure thread synchronization within a block
}