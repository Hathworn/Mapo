#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexCopy(float *res, float *src, long* res_stride, float *index, long res_nDim, int dim, long idx_size, long src_size, long size_dim)
{
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    long flat_size = src_size / idx_size;

    if (thread_idx < flat_size)
    {
        // Precompute multipliers to optimize targetIdx and resIdx calculation
        long coeff, leftover, stride_d;
        int targetIdx, resIdx;

        for (int i = 0; i < idx_size; i++)
        {
            leftover = thread_idx;
            targetIdx = 0;
            resIdx = 0;

            for (int d = 0; d < res_nDim; d++)
            {
                stride_d = (d < dim) ? (res_stride[d] / size_dim) : res_stride[d];
                coeff = leftover / stride_d;
                leftover -= coeff * stride_d;

                if (d != dim)
                {
                    targetIdx += coeff * stride_d * ((d < dim) ? idx_size : 1);
                    resIdx += coeff * res_stride[d];
                }
            }
            res[resIdx + ((int)(index[i]) - 1) * res_stride[dim]] = src[targetIdx + i * res_stride[dim]];
        }
    }
}