#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_addcdiv(float *data, float value, float *src1, float *src2, long size)
{
    // Calculate the global thread index
    long idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process each element in a strided manner to improve memory coalescing
    for (long k = idx; k < size; k += blockDim.x * gridDim.x) {
        data[k] += value * src1[k] / src2[k];
    }
}