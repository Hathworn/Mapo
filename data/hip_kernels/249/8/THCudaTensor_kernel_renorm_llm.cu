#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_renorm(float *data, const float value, const long size, const float maxnorm)
{
    __shared__ float buffer[32];
    long tx = threadIdx.x;
    long bx = blockIdx.x;
    long step = blockDim.x;
    float *row = data + size * bx;

    buffer[tx] = 0.0f;

    // Calculate norm of axis in parallel
    for (long i = tx; i < size; i += step)
    {
        buffer[tx] += powf(fabsf(row[i]), value);
    }

    // Reduction using parallel reduction
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
    {
        __syncthreads();
        if (tx < stride)
        {
            buffer[tx] += buffer[tx + stride];
        }
    }

    // Clip norms after complete reduction
    __syncthreads();
    if (tx == 0)
    {
        float norm = powf(buffer[0], 1.0f / value);
        if (norm > maxnorm)
        {
            norm = maxnorm / (norm + 1e-7f);
            buffer[0] = norm; // Store scaling factor in shared buffer
        }
    }

    // Broadcast scaling factor and renormalize
    __syncthreads();
    if (buffer[0] != 1.0f)
    {
        for (long i = tx; i < size; i += step)
        {
            row[i] *= buffer[0];
        }
    }
}