#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexSelect(float *tensor, float *src, long* src_stride, float *index, long src_nDim, int dim, long idx_size, long tensor_size, long size_dim)
{
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    long flat_size = tensor_size / idx_size;

    if (thread_idx < flat_size)
    {
        for (int i = 0; i < idx_size; i++)
        {
            int leftover = thread_idx;
            int targetIdx = 0;
            int srcIdx = 0;

            // Use loop unrolling to reduce loop overhead
            long stride_d, coeff_d;
            for (int d = 0; d < dim; d++)
            {
                stride_d = src_stride[d] / size_dim;
                coeff_d = leftover / stride_d;
                leftover -= coeff_d * stride_d;
                targetIdx += coeff_d * stride_d * idx_size;
                srcIdx += coeff_d * src_stride[d];
            }
            for (int d = dim + 1; d < src_nDim; d++)
            {
                coeff_d = leftover / src_stride[d];
                leftover -= coeff_d * src_stride[d];
                targetIdx += coeff_d * src_stride[d];
                srcIdx += coeff_d * src_stride[d];
            }
            
            // Simplify index calculation
            int index_val = (int)(index[i]) - 1; 
            tensor[targetIdx + i * src_stride[dim]] = src[srcIdx + index_val * src_stride[dim]];
        }
    }
}