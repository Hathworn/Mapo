#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_indexFill(float *tensor, long* stride, float *index, long src_nDim, int dim, long idx_size, long tensor_size, long size_dim, float val)
{
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    long flat_size = tensor_size / idx_size;

    if (thread_idx < flat_size)
    {
        long coeff = 0;
        int srcIdx = 0;
        int leftover = thread_idx;

        for (int d = 0; d < src_nDim; d++)
        {
            if (d < dim)
            {
                // Optimize stride calculations by precomputing size_dim division
                long strideDiv = stride[d] / size_dim;
                coeff = leftover / strideDiv;
                leftover -= coeff * strideDiv;
                srcIdx += coeff * stride[d];
            }
            else if (d > dim)
            {
                coeff = leftover / stride[d];
                leftover -= coeff * stride[d];
                srcIdx += coeff * stride[d];
            }
        }

        // Cache index calculation outside the loop
        int indexStride = (int)((index[thread_idx % idx_size]) - 1) * stride[dim];
        tensor[srcIdx + indexStride] = val;
    }
}