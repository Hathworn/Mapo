#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function with thread index calculation optimization
__global__ void THCudaTensor_kernel_addcmul(float *data, float value, float *src1, float *src2, long size)
{
    long k = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified thread index calculation

    if (k < size)
    {
        data[k] += value * src1[k] * src2[k];
    }
}