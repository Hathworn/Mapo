#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int f() { return 21; }

__device__ void calculateZ(int* result, int a0, int a1, int a2, int a3, int a4, int a5, int a6, int a7, int a8, int a9, int a10, int a11, int a12, int a13, int a14, int a15, int x, int modulus)
{
    // Optimized Horner's method for polynomial evaluation
    int res = a15;
    res = (res * x + a14) % modulus;
    res = (res * x + a13) % modulus;
    res = (res * x + a12) % modulus;
    res = (res * x + a11) % modulus;
    res = (res * x + a10) % modulus;
    res = (res * x + a9)  % modulus;
    res = (res * x + a8)  % modulus;
    res = (res * x + a7)  % modulus;
    res = (res * x + a6)  % modulus;
    res = (res * x + a5)  % modulus;
    res = (res * x + a4)  % modulus;
    res = (res * x + a3)  % modulus;
    res = (res * x + a2)  % modulus;
    res = (res * x + a1)  % modulus;
    res = (res * x + a0)  % modulus;
    *result = res;
}

__global__ void calcPoly16()
{
    int f_base = threadIdx.z; // is 0-4
    int a1  = threadIdx.y;
    int a2  = threadIdx.x;

    int a3  =  blockIdx.z       & 0xF;
    int a4  = (blockIdx.z >> 4) & 0xF;
    int a5  = (blockIdx.z >> 8) & 0xF;

    int a6  =  blockIdx.y       & 0xF;
    int a7  = (blockIdx.y >> 4) & 0xF;
    int a8  = (blockIdx.y >> 8) & 0xF;

    int a9  =   blockIdx.x        & 0xF;
    int a10 = (blockIdx.x >> 4)  & 0xF;
    int a11 = (blockIdx.x >> 8)  & 0xF;
    int a12 = (blockIdx.x >> 12) & 0xF;
    int a13 = (blockIdx.x >> 16) & 0xF;
    int a14 = (blockIdx.x >> 20) & 0xF;
    int a15 = (blockIdx.x >> 24) & 0xF;

    int MOD = 17; // Compiler seems automatically optimized % 16 to & 0xF

    int a0 = 4*f_base;

    int Y[10] = {0};  // Array for storing intermediate results

    for(int x = 0; x < 8; x++)
    {
        int result = 0;
        calculateZ(&result, a0, a1, a2, a3, a4, a5, a6, a7, a8, a9, a10, a11, a12, a13, a14, a15, x, MOD);
        Y[0] |= (1 << ((result + 0) % MOD));
        Y[1] |= (1 << ((result + 1) % MOD));
        Y[2] |= (1 << ((result + 2) % MOD));
        Y[3] |= (1 << ((result + 3) % MOD));
    }

    for (int idx_fg = 0; idx_fg < 4; idx_fg++)
    {
        if (Y[idx_fg] == 0b0011001101010011)
        {
            for(int x = 8; x < 16; x++)
            {
                int result = 0;
                calculateZ(&result, a0 + idx_fg, a1, a2, a3, a4, a5, a6, a7, a8, a9, a10, a11, a12, a13, a14, a15, x, MOD);
                Y[4 + idx_fg] |= (1 << (result % MOD));
            }

            if (Y[4+idx_fg] == 0b1100110010101100)
            {
                int res[16];
                for(int tmpi = 0; tmpi < 16; tmpi++)
                {
                    calculateZ(&res[tmpi], a0 + idx_fg, a1, a2, a3, a4, a5, a6, a7, a8, a9, a10, a11, a12, a13, a14, a15, tmpi, MOD);
                }

                printf("a=[%2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d, %2d,], res=[%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,%2d,] Y=[%d %d %d %d %d %d %d %d %d %d]\n",
                       a0 + idx_fg, a1, a2, a3, a4, a5, a6, a7, a8, a9, a10, a11, a12, a13, a14, a15, 
                       res[0], res[1], res[2], res[3], res[4], res[5], res[6], res[7], res[8], res[9], res[10], res[11], res[12], res[13], res[14], res[15],
                       Y[0], Y[1], Y[2], Y[3], Y[4], Y[5], Y[6], Y[7], Y[8], Y[9]);
            }
        }
    }
}