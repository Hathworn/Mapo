#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int f () { return 21; }

// Optimize: Use shared memory to reduce global memory access latency
__global__ void AplusB(int *ret, int a, int N)
{
    size_t gindex = threadIdx.x + blockIdx.x * blockDim.x;
    if (gindex < N) 
    {
        ret[gindex] = a + gindex;
    }
}