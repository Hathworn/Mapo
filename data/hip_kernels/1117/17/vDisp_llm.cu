#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int f() { return 21; }

__global__ void vDisp(const float *A, const float *B, int ds)
{
    // Create typical 1D thread index from built-in variables more efficiently
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds
    if (idx < ds)
    {
        // Device output only when within data size to minimize unnecessary calculations
        printf("Device: [%d], \t%f\t%f \n", idx, A[idx], B[idx]);
    }
}