#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initMemory(size_t position, size_t* array)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Calculate the global index once
    size_t globalIdx = position + idx;

    // Use global index to access memory
    array[globalIdx] = idx;
}