#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int f () { return 21; }
__global__ void sample()
{
    int a = blockIdx.x;
    int b = blockIdx.y;
    int c = threadIdx.x;
    double x = 1;

    // Optimize pow(0.0, x) to 0 and pow(x, 2.0) to x * x
    double result = a + b * x + c * (x * x);

    if(result == 10)
        printf("a=%d, b=%d, c=%d\n", a, b, c);
}