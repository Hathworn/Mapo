#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kArgMinColumnwise(float* mat, float* target, unsigned int width, unsigned int height) {
    __shared__ float min_vals[32];
    __shared__ unsigned int min_args[32];
    float cur_min = 2e38;
    unsigned int cur_arg = 0;
    float val = 0;

    // Loop stride optimization to avoid warp divergence
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
        val = mat[i * width + blockIdx.x];

        // Perform comparison and update min
        if (val < cur_min) {
            cur_min = val;
            cur_arg = i;
        }
    }

    min_vals[threadIdx.x] = cur_min;
    min_args[threadIdx.x] = cur_arg;

    __syncthreads();

    // Reduce within the warp
    if (threadIdx.x < 16) {
        for (int offset = 16; offset > 0; offset /= 2) {
            if (threadIdx.x + offset < 32 && min_vals[threadIdx.x + offset] < min_vals[threadIdx.x]) {
                min_vals[threadIdx.x] = min_vals[threadIdx.x + offset];
                min_args[threadIdx.x] = min_args[threadIdx.x + offset];
            }
        }
    }

    // Only the first thread writes the result
    if (threadIdx.x == 0) {
        target[blockIdx.x] = min_args[0];
    }
}