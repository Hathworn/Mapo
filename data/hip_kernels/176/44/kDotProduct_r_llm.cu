#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;  // Local accumulator
    // Loop unrolling for more efficient memory access
    for (int i = eidx; i < numElements; i += gridDim.x * DP_BLOCKSIZE) {
        sum += a[i] * b[i];
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Reduce using warp-level operations
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        if(threadIdx.x + 32 < DP_BLOCKSIZE) mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        if(threadIdx.x + 16 < DP_BLOCKSIZE) mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        if(threadIdx.x + 8 < DP_BLOCKSIZE) mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        if(threadIdx.x + 4 < DP_BLOCKSIZE) mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        if(threadIdx.x + 2 < DP_BLOCKSIZE) mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        if(threadIdx.x + 1 < DP_BLOCKSIZE) mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}