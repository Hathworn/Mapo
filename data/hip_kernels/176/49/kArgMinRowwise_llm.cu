#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kArgMinRowwise(float* mat, float* target, unsigned int width, unsigned int height) {
    __shared__ float min_vals[32];
    __shared__ unsigned int min_args[32];
    
    // Initialize shared memory with maximum values
    min_vals[threadIdx.x] = 2e38;
    min_args[threadIdx.x] = 0;

    float cur_min = 2e38;
    unsigned int cur_arg = 0;
    float val = 0;

    // Process elements in the row assigned to the block
    for (unsigned int i = threadIdx.x; i < width; i += blockDim.x) {
        val = mat[blockIdx.x * width + i];

        if (val < cur_min) {
            cur_min = val;
            cur_arg = i;
        }
    }

    // Atomically update shared memory to ensure thread safety
    atomicMin(reinterpret_cast<unsigned int*>(&min_vals[threadIdx.x]), __float_as_uint(cur_min));
    atomicMin(&min_args[threadIdx.x], cur_arg);

    __syncthreads();

    // Use one thread to determine the global minimum argument
    if (threadIdx.x == 0) {
        cur_min = 2e38;
        cur_arg = 0;

        for (unsigned int i = 0; i < blockDim.x; i++) {
            if (min_vals[i] < cur_min) {
                cur_min = min_vals[i];
                cur_arg = min_args[i];
            }
        }

        target[blockIdx.x] = cur_arg;
    }
}