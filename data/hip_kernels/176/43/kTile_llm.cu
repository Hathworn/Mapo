#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate unique thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total number of threads
    const int numThreads = blockDim.x * gridDim.x;

    // Loop over the output elements, step by the number of threads
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Pre-compute indices for better memory access pattern
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Ensure coalesced memory access by reading from global memory with calculated indices
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}