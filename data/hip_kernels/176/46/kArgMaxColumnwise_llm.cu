#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kArgMaxColumnwise(float* mat, float* target, unsigned int width, unsigned int height) {
    __shared__ float max_vals[32];
    __shared__ unsigned int max_args[32];
    
    float cur_max = -2e38;
    unsigned int cur_arg = 0;
    float val = 0;
    
    // Use a loop stride to allow arbitrary block size 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
        val = mat[i * width + blockIdx.x];
        
        if (val > cur_max) {
            cur_max = val;
            cur_arg = i;
        }
    }
    
    max_vals[threadIdx.x] = cur_max;
    max_args[threadIdx.x] = cur_arg;
    
    __syncthreads();
    
    // Reduce within block to find the maximum
    if (threadIdx.x == 0) {
        cur_max = -2e38;
        cur_arg = 0;
        
        for (unsigned int i = 0; i < blockDim.x; i++) {
            if (max_vals[i] > cur_max) {
                cur_max = max_vals[i];
                cur_arg = max_args[i];
            }
        }
        
        target[blockIdx.x] = cur_arg;
    }
}