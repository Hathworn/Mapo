#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Use a more efficient register variable for state pointer
    hiprandState localState;
    hiprand_init(seed, tidx, 0, &localState);
    state[tidx] = localState;
}