#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Early exit if the label probability does not equal the max probability
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 1; // Assume 1 max probability initially
            for (int i = 0; i < numOut; i++) {
                // Compare all except current label which is already considered
                if (i != label) {
                    numMax += probs[i * numCases + tx] == maxp;
                }
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}