#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultiSoftmaxCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, float* top5Probs, const int numCases, const int numOut, const int setSize) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Use shared memory to reduce access time
        extern __shared__ float s_probs[];
        for (int i = threadIdx.x; i < numOut; i += blockDim.x) {
            s_probs[i] = probs[i * numCases + tx];
        }
        __syncthreads();

        // Unroll the loop for better performance
        int numBiggerProbs = 0, numEqualsProbs = 0;
        for (int i = 0; i < numOut; i++) {
            float prob = s_probs[i];
            numBiggerProbs += prob > labelp;
            numEqualsProbs += prob == labelp;
        }

        const int slotsLeft = setSize - numBiggerProbs;
        top5Probs[tx] = slotsLeft <= 0 ? 0.0f : (numEqualsProbs <= slotsLeft ? 1.0f : float(slotsLeft) / numEqualsProbs);
        correctProbs[tx] = labelp != maxp ? 0.0f : 1.0f / float(numEqualsProbs);
    }
}