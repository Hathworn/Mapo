#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kArgMaxRowwise(float* mat, float* target, unsigned int width, unsigned int height) {
    __shared__ float max_vals[32];
    __shared__ unsigned int max_args[32];
    
    // Initialize cur_max with a low value for each thread
    float cur_max = -2e38;
    unsigned int cur_arg = 0;
    
    // Optimize loop by using stride to cover the entire width in a coalesced manner
    for (unsigned int i = threadIdx.x; i < width; i += blockDim.x) {
        float val = mat[blockIdx.x * width + i];
        if (val > cur_max) {
            cur_max = val;
            cur_arg = i;
        }
    }
    
    // Store results in shared memory
    max_vals[threadIdx.x] = cur_max;
    max_args[threadIdx.x] = cur_arg;
    __syncthreads();
    
    // Perform reduction using sequential addressing
    if (threadIdx.x == 0) {
        cur_max = max_vals[0];
        cur_arg = max_args[0];
        for (unsigned int i = 1; i < blockDim.x; i++) {
            if (max_vals[i] > cur_max) {
                cur_max = max_vals[i];
                cur_arg = max_args[i];
            }
        }
        target[blockIdx.x] = cur_arg;
    }
}