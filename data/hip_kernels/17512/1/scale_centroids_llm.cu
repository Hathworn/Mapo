#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_centroids(int d, int k, int* counts, double* centroids) {
    int global_id_x = threadIdx.x + blockIdx.x * blockDim.x;
    int global_id_y = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Use shared memory for counts to reduce global memory access
    __shared__ int shared_counts[1024]; // Adjust size based on maximum possible 'k'
    
    if (global_id_y < k && threadIdx.x == 0) {
        shared_counts[global_id_y] = counts[global_id_y];
    }
    __syncthreads(); // Ensure all threads have loaded the data
    
    if (global_id_x < d && global_id_y < k) {
        int count = shared_counts[global_id_y];
        // To avoid introducing divide by zero errors
        // If a centroid has no weight, we'll do no normalization
        // This will keep its coordinates defined.
        if (count < 1) {
            count = 1;
        }
        double scale = 1.0 / double(count);
        centroids[global_id_x + d * global_id_y] *= scale;
    }
}