#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void self_dots(int n, int d, double* data, double* dots) {
    // Use shared memory for partial sums
    extern __shared__ double shared_data[];
    double accumulator = 0;
    int global_id = blockDim.x * blockIdx.x + threadIdx.x;

    if (global_id < n) {
        for (int i = threadIdx.x; i < d; i += blockDim.x) {
            double value = data[i + global_id * d];
            accumulator += value * value;
        }
        // Store the result in shared memory
        shared_data[threadIdx.x] = accumulator;

        // Synchronize to ensure all additions are complete
        __syncthreads();

        // Perform reduction in shared memory
        for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
            if (threadIdx.x < offset) {
                shared_data[threadIdx.x] += shared_data[threadIdx.x + offset];
            }
            __syncthreads();
        }
        // Write the result to the global memory
        if (threadIdx.x == 0) {
            dots[blockIdx.x] = shared_data[0];
        }
    }
}