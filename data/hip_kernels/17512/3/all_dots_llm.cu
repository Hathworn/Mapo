#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void all_dots(int n, int k, double* data_dots, double* centroid_dots, double* dots) {
    __shared__ double local_data_dots[32];
    __shared__ double local_centroid_dots[32];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int data_index = tx + blockIdx.x * blockDim.x;
    int centroid_index = ty + blockIdx.y * blockDim.y;

    // Load data_dots into shared memory by the first warp
    if (data_index < n && ty == 0) {
        local_data_dots[tx] = data_dots[data_index];
    }
    // Load centroid_dots into shared memory by the second warp
    if (centroid_index < k && ty == 1) {
        local_centroid_dots[tx] = centroid_dots[centroid_index];
    }

    __syncthreads();

    // Calculate dots using shared memory
    if (data_index < n && centroid_index < k) {
        dots[data_index + centroid_index * n] = local_data_dots[tx] + local_centroid_dots[ty];
    }
}