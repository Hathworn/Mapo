#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cube(float * d_out, float * d_in) {
    // Use blockIdx and blockDim for better scalability
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float f = d_in[index];
    d_out[index] = f * f * f;
}