#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    // Calculate the starting address for the current block
    unsigned int* orig_ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* ptr = orig_ptr;

    // Early exit if the starting address is beyond the end
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Calculate the end address for this block
    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);

    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Initialize the starting address with the pattern
    *ptr = pattern;

    // Loop through and set memory locations as per pattern
    while (ptr < block_end) {
        ptr = (unsigned int*)(((unsigned long)orig_ptr) | mask);

        // If computed address equals the original, shift the mask and continue
        if (ptr == orig_ptr) {
            mask <<= 1;
            continue;
        }

        // Break if the computed address is beyond the block end
        if (ptr >= block_end) {
            break;
        }

        // Set the memory location with the current pattern
        *ptr = pattern;

        // Update the pattern and mask for the next iteration
        pattern <<= 1;
        mask <<= 1;
    }
}