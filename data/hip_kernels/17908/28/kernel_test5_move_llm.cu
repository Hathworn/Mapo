#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    // Calculate initial pointer for this block
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Check if this block's data is out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Precompute values for loop counts
    unsigned int half_count = BLOCKSIZE / sizeof(unsigned int) / 2;
    unsigned int* ptr_mid = ptr + half_count;

    // Use shared memory for improved access times
    __shared__ unsigned int shared_mem[BLOCKSIZE / sizeof(unsigned int)];
    
    // Load data into shared memory
    if (threadIdx.x < half_count) {
        shared_mem[threadIdx.x] = ptr[threadIdx.x];
        shared_mem[threadIdx.x + half_count] = ptr_mid[threadIdx.x];
    }
    __syncthreads();

    // Move data from the first half to the second half
    if (threadIdx.x < half_count - 8) {
        ptr[threadIdx.x + 8] = shared_mem[threadIdx.x + half_count];
    }
    if (threadIdx.x < 8) {
        ptr[threadIdx.x] = shared_mem[half_count - 8 + threadIdx.x];
    }
}