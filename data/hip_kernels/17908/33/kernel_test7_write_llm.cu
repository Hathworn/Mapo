#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    unsigned int idx = blockIdx.x * BLOCKSIZE / sizeof(unsigned int) + threadIdx.x;  // Optimize index calculation
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* start_ptr = (unsigned int*)_start_ptr;

    __shared__ unsigned int shared_data[BLOCKSIZE / sizeof(unsigned int)];  // Use shared memory for performance

    if ((unsigned char*)(ptr + idx) >= end_ptr) {
        return;  // Ensure out-of-bounds prevention
    }

    shared_data[threadIdx.x] = start_ptr[threadIdx.x];  // Load data into shared memory

    __syncthreads();  // Synchronize threads before writing to global memory

    ptr[idx] = shared_data[threadIdx.x];  // Write data from shared memory to global memory

    return;
}