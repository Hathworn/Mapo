#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    // Calculate a single global index for each thread in the grid
    unsigned int globalIndex = blockIdx.x * BLOCKSIZE / sizeof(unsigned int) + threadIdx.x;
    unsigned int* ptr = (unsigned int*)_ptr;

    // Ensure the thread has valid memory access
    if ((ptr + globalIndex) < (unsigned int*)end_ptr) {
        // Write pattern using individual threads
        ptr[globalIndex] = pattern;
    }
}