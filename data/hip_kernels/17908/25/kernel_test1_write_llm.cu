#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Early return if block's starting point is beyond end_ptr
    if (ptr >= (unsigned long*) end_ptr) {
        return;
    }

    // Use threadIdx to reduce unnecessary iterations
    unsigned int tid = threadIdx.x; 
    unsigned int stride = blockDim.x;

    for (unsigned int i = tid; i < BLOCKSIZE / sizeof(unsigned long); i += stride) {
        ptr[i] = (unsigned long) &ptr[i];
    }

    return;
}