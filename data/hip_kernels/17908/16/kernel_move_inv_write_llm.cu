#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    // Calculate the starting index for the current thread block
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Check if the starting index exceeds the end pointer
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Use a loop unrolling technique to reduce loop overhead
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        ptr[i] = pattern;
    }
    return;
}