#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Exit early if out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int k = offset;
    unsigned int pat = pattern;

    // Use shared memory for faster access
    __shared__ unsigned int shared_pat[BLOCKSIZE / sizeof(unsigned int)];

    // Iterate using the thread index to process multiple elements per thread
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        shared_pat[i] = pat;
        k++;
        if (k >= 32) {
            k = 0;
            pat = lb;
        } else {
            pat = pat << 1;
            pat |= sval;
        }
    }

    // Synchronize threads to ensure all shared memory writes are done
    __syncthreads();

    // Write back to global memory
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        ptr[i] = shared_pat[i];
    }
}