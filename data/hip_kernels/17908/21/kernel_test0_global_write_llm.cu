#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int* orig_ptr = ptr;

    unsigned int pattern = 1;
    unsigned long mask = 4;

    *ptr = pattern;

    // Precompute pattern shift count
    int shift_count = 0;

    while(ptr < end_ptr){

        ptr = (unsigned int*)(((unsigned long)orig_ptr) | mask);
        if (ptr == orig_ptr){
            mask = mask << 1;
            shift_count++;
            continue;
        }
        if (ptr >= end_ptr){
            break;
        }

        *ptr = pattern;

        // Use precomputed shift count
        pattern = 1 << shift_count++;
        mask = mask << 1;
    }
    return;
}