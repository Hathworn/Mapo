#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_init(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Early exit if block is beyond end_ptr
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }
    
    unsigned int p1 = 1;
    // Loop unrolling to reduce loop overhead
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i += 16) {
        unsigned int p2 = ~p1;
        
        #pragma unroll
        for (unsigned int j = 0; j < 16; j += 4) {
            ptr[i + j] = p1;
            ptr[i + j + 1] = p1;
            ptr[i + j + 2] = p2;
            ptr[i + j + 3] = p2;
        }

        // Optimize bit shifting by using a mask
        p1 = (p1 << 1) | (p1 >> (31));
    }
}