#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }
    
    // Use shared memory for faster access
    __shared__ unsigned int shared_mem[BLOCKSIZE/sizeof(unsigned int)];

    // Initialize shared memory
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE/sizeof(unsigned int); i += blockDim.x) {
        shared_mem[i] = (i % MOD_SZ == offset) ? p1 : p2;
    }

    // Synchronize threads within block to ensure all threads have written to shared memory
    __syncthreads();

    // Copy from shared memory to global memory
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE/sizeof(unsigned int); i += blockDim.x) {
        ptr[i] = shared_mem[i];
    }
    
    return;
}