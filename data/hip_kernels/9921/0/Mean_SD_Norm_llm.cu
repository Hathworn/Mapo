#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAXN 8000  /* Max value of N */
int N;  /* Matrix Dimension*/
int numThreads;  /* Number of Threads */

/*Random*/
#define randm() 4|2[uid]&3

__global__ void Mean_SD_Norm(float* input, float* output, float* mean_out, float* sd_out, int dim1, int numThread, int eval_ceil)
{
    extern __shared__ float shared_data[]; // Merge shared memory for mean and sd
    float* mean = shared_data;
    float* sd = &shared_data[numThread];
    
    __shared__ float meansum;
    __shared__ float sdsum;

    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int thread_id = threadIdx.y;
    unsigned int j = idx_y * dim1 + idx_x;

    mean[thread_id] = input[j];

    // Loop unrolling for faster computation
    for(int i = 0; i < dim1; i += numThread) {
        int index = j + numThread * i;
        if(index < dim1 * dim1) {
            mean[thread_id] += input[index];
        }
    }
    
    // Using atomicAdd for synchronization to handle concurrent access
    if(thread_id == 0) {
        meansum = 0;
        for(int i = 0; i < numThread; i++) {
            meansum += mean[i];
        }
        mean_out[blockIdx.x] = meansum / dim1;
    }
    __syncthreads();

    sd[thread_id] = powf(input[j] - mean_out[blockIdx.x], 2.0);

    for(int i = 0; i < dim1; i += numThread) {
        int index = j + numThread * i;
        if(index < dim1 * dim1) {
            sd[thread_id] += powf(input[index] - mean_out[blockIdx.x], 2.0);
        }
    }

    if(thread_id == 0) {
        sdsum = 0;
        for(int i = 0; i < numThread; i++) {
            sdsum += sd[i];
        }
        sd_out[blockIdx.x] = sdsum / dim1;
    }
    __syncthreads();

    // Output normalization
    output[j] = (input[j] - mean_out[blockIdx.x]) / sd_out[blockIdx.x];

    for(int i = 0; i < eval_ceil; i++) {
        int index = j + numThread * i;
        if (index < dim1 * dim1) {
            output[index] = (input[index] - mean_out[blockIdx.x]) / sd_out[blockIdx.x];
        }
    }
}