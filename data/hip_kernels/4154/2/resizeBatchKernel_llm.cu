#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void resizeBatchKernel(const uint8_t *p_Src, int nSrcPitch, int nSrcHeight, uint8_t *p_dst, int nDstWidth, int nDstHeight) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uchar3 rgb;
    // Pre-calculate scaling factors outside the condition to avoid redundant division
    float yScaleFactor = static_cast<float>(nSrcHeight) / nDstHeight;
    float xScaleFactor = static_cast<float>(nSrcPitch) / nDstWidth;
    
    if (tid < nDstWidth && tidd < nDstHeight) {
        // Scale source coordinates using pre-calculated factors
        int j = static_cast<int>(tidd * yScaleFactor) * nSrcPitch;
        int k = static_cast<int>(tid * xScaleFactor) * 3; // Adjust by 3 for RGB
        
        // Load pixel data
        rgb.x = p_Src[j + k + 0];
        rgb.y = p_Src[j + k + 1];
        rgb.z = p_Src[j + k + 2];
        
        // Calculate destination index
        int destIndex = (tidd * nDstWidth + tid) * 3;
        
        // Store pixel data
        p_dst[destIndex + 0] = rgb.x;
        p_dst[destIndex + 1] = rgb.y;
        p_dst[destIndex + 2] = rgb.z;
    }
}