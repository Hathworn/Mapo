#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void convertPToRGBKernel(const uint16_t *dpSrc, uint8_t *tt1, int nSrcWidth, int nDstWidth, int nDstHeight, int *lookupTable) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t v0, y0, u0, y1;
    int nDstH = nDstHeight;
    int nDstW = nSrcWidth / 2;

    // Ensure thread indices are within bounds
    if (tid < nDstW && tidd < nDstH) {
        int k = tid * 2;
        int j = tidd * nSrcWidth;

        // Load Y and U, V components; avoid recalculation
        y0 = static_cast<uint32_t>(dpSrc[j + k]) * 1000;
        y1 = static_cast<uint32_t>(dpSrc[j + k + 1]) * 1000;

        k = tid;
        j = tidd * nSrcWidth / 2 + nDstHeight * nSrcWidth;
        u0 = static_cast<uint32_t>(dpSrc[j + k]);

        j = tidd * nSrcWidth / 2 + nDstHeight * nSrcWidth * 3 / 2;
        v0 = static_cast<uint32_t>(dpSrc[j + k]);

        // Calculate color components only once
        int r = 1407 * v0 - 720384, g = 716 * v0 + 345 * u0 - 543232, b = 1779 * u0 - 910848;

        k = tid * 6;
        j = tidd * nDstWidth * 3;

        // Use shared temporary variables for conversion
        uint16_t tt0 = (y0 + r) / 1000;
        uint16_t tt1_tmp = (y0 - g) / 1000;
        uint16_t tt2 = (y0 + b) / 1000;
        uint16_t tt3 = (y1 + r) / 1000;
        uint16_t tt4 = (y1 - g) / 1000;
        uint16_t tt5 = (y1 + b) / 1000;

        // Perform look-up table transformation
        tt1[j + k] = lookupTable[tt0];
        tt1[j + k + 1] = lookupTable[tt1_tmp];
        tt1[j + k + 2] = lookupTable[tt2];
        tt1[j + k + 3] = lookupTable[tt3];
        tt1[j + k + 4] = lookupTable[tt4];
        tt1[j + k + 5] = lookupTable[tt5];
    }
}