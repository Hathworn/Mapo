#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized Kernel Function
__global__ static void resizeBatchKernel(const uint16_t *p_Src, int nSrcPitch, int nSrcHeight, uint8_t *dpDst0, uint8_t *dpDst1, uint8_t *dpDst2, int nDstWidth, int nDstHeight, int *lookupTable_cuda) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    int scale = nSrcHeight / nDstHeight;
    int nDstH = nDstHeight;
    int nDstW = (scale == 6) ? nDstWidth / 2 : nDstWidth / 6;

    if (tid < nDstW && tidd < nDstH) {
        int j = tidd * nSrcPitch * scale;
        int k = tid * ((scale == 6) ? 16 : 32);

        uint32_t v0, y0, u0, y1, u1, v1, y2, y3, y4, y5;
        
        uint4 pF;
        // Common variable loading and calculation
        pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);

        if (scale == 4 || scale == 2) {
            pF.w = (uint32_t)p_Src[j + k + 6];
            v0 = (pF.x & 0x3FF00000) >> 20;
            y0 = (pF.x & 0x000FFC00) >> 10;
            u0 = pF.x & 0x000003FF;
            y1 = pF.w & 0x000003FF;

            // Further processing based on scale
            if (scale == 4) {
                pF.y = (uint32_t)p_Src[j + k + 10] + ((uint32_t)p_Src[j + k + 11] << 16);
                pF.z = (uint32_t)p_Src[j + k + 12];

                y2 = (pF.y & 0x3FF00000) >> 20;
                u1 = (pF.y & 0x000FFC00) >> 10;
                v1 = pF.z & 0x000003FF;

                pF.x = (uint32_t)p_Src[j + k + 16] + ((uint32_t)p_Src[j + k + 17] << 16);
                pF.z = ((uint32_t)p_Src[j + k + 21] << 16);
                pF.w = (uint32_t)p_Src[j + k + 22] + ((uint32_t)p_Src[j + k + 23] << 16);

                y3 = (pF.x & 0x000FFC00) >> 10;
                uint32_t u2 = (pF.z & 0x3FF00000) >> 20;
                uint32_t v2 = (pF.w & 0x000FFC00) >> 10;
                y4 = pF.w & 0x000003FF;
                pF.y = ((uint32_t)p_Src[j + k + 27] << 16);
                y5 = (pF.y & 0x3FF00000) >> 20;

                k = tid * 6;
                j = tidd * nDstWidth;
                dpDst0[j + k + 0] = lookupTable_cuda[y0];
                dpDst0[j + k + 1] = lookupTable_cuda[y1];
                dpDst0[j + k + 2] = lookupTable_cuda[y2];
                dpDst0[j + k + 3] = lookupTable_cuda[y3];
                dpDst0[j + k + 4] = lookupTable_cuda[y4];
                dpDst0[j + k + 5] = lookupTable_cuda[y5];
                
                k = tid * 3;
                j = tidd * nDstWidth / 2;
                dpDst1[j + k + 0] = lookupTable_cuda[u0];
                dpDst1[j + k + 1] = lookupTable_cuda[u1];
                dpDst1[j + k + 2] = lookupTable_cuda[u2];
                dpDst2[j + k + 0] = lookupTable_cuda[v0];
                dpDst2[j + k + 1] = lookupTable_cuda[v1];
                dpDst2[j + k + 2] = lookupTable_cuda[v2];
            } else { // scale == 2
                pF.y = ((uint32_t)p_Src[j + k + 3] << 16);
                pF.z = ((uint32_t)p_Src[j + k + 5] << 16);
                v1 = (pF.w & 0x000FFC00) >> 10;
                y2 = pF.w & 0x000003FF;
                y1 = (pF.y & 0x3FF00000) >> 20;
                u1 = (pF.z & 0x3FF00000) >> 20;

                pF.x = (uint32_t)p_Src[j + k + 8] + ((uint32_t)p_Src[j + k + 9] << 16);
                y3 = (pF.x & 0x000FFC00) >> 10;
                y4 = (uint32_t)(((uint32_t)p_Src[j + k + 10] + ((uint32_t)p_Src[j + k + 11] << 16) & 0x3FF00000) >> 20);
                uint32_t u2 = (pF.x & 0x000FFC00) >> 10;
                uint32_t v2 = ((uint32_t)p_Src[j + k + 12] & 0x000003FF);
                y5 = ((uint32_t)p_Src[j + k + 14] & 0x000003FF);

                k = tid * 6;
                j = tidd * nDstWidth;
                dpDst0[j + k + 0] = lookupTable_cuda[y0];
                dpDst0[j + k + 1] = lookupTable_cuda[y1];
                dpDst0[j + k + 2] = lookupTable_cuda[y2];
                dpDst0[j + k + 3] = lookupTable_cuda[y3];
                dpDst0[j + k + 4] = lookupTable_cuda[y4];
                dpDst0[j + k + 5] = lookupTable_cuda[y5];
                
                k = tid * 3;
                j = tidd * nDstWidth / 2;
                dpDst1[j + k + 0] = lookupTable_cuda[u0];
                dpDst1[j + k + 1] = lookupTable_cuda[u1];
                dpDst1[j + k + 2] = lookupTable_cuda[u2];
                dpDst2[j + k + 0] = lookupTable_cuda[v0];
                dpDst2[j + k + 1] = lookupTable_cuda[v1];
                dpDst2[j + k + 2] = lookupTable_cuda[v2];
            }
        } else { // scale == 6
            v0 = (pF.x & 0x3FF00000) >> 20;
            y0 = (pF.x & 0x000FFC00) >> 10;
            u0 = pF.x & 0x000003FF;

            pF.x = (uint32_t)p_Src[j + k + 8] + ((uint32_t)p_Src[j + k + 9] << 16);
            y1 = (pF.x & 0x000FFC00) >> 10;

            k = tid * 2;
            j = tidd * nDstWidth;
            dpDst0[j + k + 0] = lookupTable_cuda[y0];
            dpDst0[j + k + 1] = lookupTable_cuda[y1];

            k = tid;
            j = tidd * nDstWidth / 2;
            dpDst1[j + k + 0] = lookupTable_cuda[u0];
            dpDst2[j + k + 1] = lookupTable_cuda[v0];
        }
    }
}