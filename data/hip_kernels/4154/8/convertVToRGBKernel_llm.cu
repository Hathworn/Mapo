#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void convertVToRGBKernel(const uint16_t *pV210, uint8_t *tt1, int nSrcWidth, int nDstWidth, int nDstHeight, int *lookupTable) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uint16_t tt[6];
    int nDstH = nDstHeight;
    int nDstW = nSrcWidth / 8;

    if (tid < nDstW && tidd < nDstH) {
        // Refactor repetitive calculations into a function for clarity and potential performance.
        auto processPixel = [&](uint32_t y, uint32_t u, uint32_t v, uint8_t *output) {
            int r = 1407 * v - 720384, g = 716 * v + 345 * u - 543232, b = 1779 * u - 910848;
            tt[0] = (y + r) / 1000;
            tt[1] = (y - g) / 1000;
            tt[2] = (y + b) / 1000;
            output[0] = lookupTable[tt[0]];
            output[1] = lookupTable[tt[1]];
            output[2] = lookupTable[tt[2]];
        };

        int j = (tidd * nSrcWidth * 9) / 4;
        int k = tid * 18;

        uint4 pF;
        int baseIdx = tidd * nSrcWidth + tid * 8;
        pF.x = (uint32_t)pV210[baseIdx + 0] + ((uint32_t)pV210[baseIdx + 1] << 16);
        pF.y = (uint32_t)pV210[baseIdx + 2] + ((uint32_t)pV210[baseIdx + 3] << 16);
        pF.z = (uint32_t)pV210[baseIdx + 4] + ((uint32_t)pV210[baseIdx + 5] << 16);
        pF.w = (uint32_t)pV210[baseIdx + 6] + ((uint32_t)pV210[baseIdx + 7] << 16);

        // Process chunks of the pixel data
        processPixel((pF.x & 0x000FFC00) >> 10 * 1000, (pF.x & 0x000003FF), (pF.x & 0x3FF00000) >> 20, &tt1[j + k + 0]);
        processPixel((pF.y & 0x000003FF) * 1000, (pF.y & 0x000FFC00) >> 10, (pF.y & 0x3FF00000) >> 20, &tt1[j + k + 3]);
        processPixel((pF.z & 0x000FFC00) >> 10 * 1000, (pF.z & 0x3FF00000) >> 20, (pF.z & 0x000003FF), &tt1[j + k + 6]);
        processPixel((pF.w & 0x000003FF) * 1000, (pF.w & 0x000FFC00) >> 10, (pF.w & 0x3FF00000) >> 20, &tt1[j + k + 9]);
        processPixel((pF.z & 0x000FFC00) >> 10 * 1000, (pF.z & 0x3FF00000) >> 20, (pF.z & 0x000003FF), &tt1[j + k + 12]);
    }
}