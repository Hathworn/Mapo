#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void convertToNppiKernel(uint16_t *dSrc, uint8_t *dDst, int nSrcWidth, int nDstWidth, int nDstHeight, int *lookupTable) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    int nDstH = nDstHeight;
    int nDstW = nSrcWidth / 8;

    // Ensure we're within bounds
    if (tid >= nDstW || tidd >= nDstH) return;

    int j = tidd * nSrcWidth;
    int k = tid * 8;
    uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
    uint16_t tt[6];
    uint4 pF;

    pF.x = (uint32_t)dSrc[j + k + 0] + ((uint32_t)dSrc[j + k + 1] << 16);
    pF.y = (uint32_t)dSrc[j + k + 2] + ((uint32_t)dSrc[j + k + 3] << 16);
    pF.z = (uint32_t)dSrc[j + k + 4] + ((uint32_t)dSrc[j + k + 5] << 16);
    pF.w = (uint32_t)dSrc[j + k + 6] + ((uint32_t)dSrc[j + k + 7] << 16);

    // Extract and compute values
    const int shift1 = 20, shift2 = 10;
    const int multFactor = 1000;
    v0 = (pF.x >> shift1) & 0x3FF;
    y0 = ((pF.x >> shift2) & 0x3FF) * multFactor;
    u0 = pF.x & 0x3FF;
    y2 = ((pF.y >> shift1) & 0x3FF) * multFactor;
    u1 = (pF.y >> shift2) & 0x3FF;
    y1 = (pF.y & 0x3FF) * multFactor;
    u2 = (pF.z >> shift1) & 0x3FF;
    y3 = ((pF.z >> shift2) & 0x3FF) * multFactor;
    v1 = pF.z & 0x3FF;
    y5 = ((pF.w >> shift1) & 0x3FF) * multFactor;
    v2 = (pF.w >> shift2) & 0x3FF;
    y4 = (pF.w & 0x3FF) * multFactor;

    // Common calculations reduction using lambdas
    auto computeRGB = [multFactor](int y, int v, int u, int &r, int &g, int &b) {
        r = 1407 * v - 720384;
        g = 716 * v + 345 * u - 543232;
        b = 1779 * u - 910848;
    };

    k = tid * 18;
    j = j * 9 / 4;
    
    int r, g, b;
    // Process and store results using a loop
    auto processAndStore = [&](int idx, int y, int v, int u) {
        computeRGB(y, v, u, r, g, b);
        tt[0] = (y + r) / multFactor;
        tt[1] = (y - g) / multFactor;
        tt[2] = (y + b) / multFactor;
        
        dDst[j + k + idx] = lookupTable[tt[0]];
        dDst[j + k + idx + 1] = lookupTable[tt[1]];
        dDst[j + k + idx + 2] = lookupTable[tt[2]];
    };

    processAndStore(0, y0, v0, u0);
    processAndStore(3, y1, v0, u0);
    processAndStore(6, y2, v1, u1);
    processAndStore(9, y3, v1, u1);
    processAndStore(12, y4, v2, u2);
    processAndStore(15, y5, v2, u2);
}