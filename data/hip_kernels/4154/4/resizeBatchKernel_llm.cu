#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void resizeBatchKernel(const uint16_t *p_Src, int nSrcPitch, int nSrcHeight, uint8_t *dpDst, int nDstWidth, int nDstHeight, int *lookupTable_cuda) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;

    uint4 pF;
    int scale = nSrcHeight / nDstHeight;

    // Using shared memory for reduced global memory access
    __shared__ uint16_t buffer[32];
    
    if (scale == 4) {
        uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
        int nDstH = nDstHeight;
        int nDstW = nDstWidth / 6;
        if (tid < nDstW && tidd < nDstH) {
            int j = tidd * nSrcPitch * scale;
            int k = tid * 32;

            // Load source data into shared memory
            buffer[threadIdx.x + 0] = p_Src[j + k + 0];
            buffer[threadIdx.x + 1] = p_Src[j + k + 1];
            buffer[threadIdx.x + 6] = p_Src[j + k + 6];
            buffer[threadIdx.x + 10] = p_Src[j + k + 10];
            buffer[threadIdx.x + 11] = p_Src[j + k + 11];
            buffer[threadIdx.x + 12] = p_Src[j + k + 12];
            buffer[threadIdx.x + 16] = p_Src[j + k + 16];
            buffer[threadIdx.x + 17] = p_Src[j + k + 17];
            buffer[threadIdx.x + 21] = p_Src[j + k + 21];
            buffer[threadIdx.x + 22] = p_Src[j + k + 22];
            buffer[threadIdx.x + 23] = p_Src[j + k + 23];
            buffer[threadIdx.x + 27] = p_Src[j + k + 27];

            __syncthreads();

            // Process data using local variables for improved register use
            pF.x = (uint32_t)buffer[0] + ((uint32_t)buffer[1] << 16);
            pF.w = (uint32_t)buffer[6];
            v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
            y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            u0 = (uint32_t)(pF.x & 0x000003FF);
            y1 = (uint32_t)(pF.w & 0x000003FF);

            pF.y = (uint32_t)buffer[10] + ((uint32_t)buffer[11] << 16);
            pF.z = (uint32_t)buffer[12];
            y2 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
            u1 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
            v1 = (uint32_t)(pF.z & 0x000003FF);

            pF.x = (uint32_t)buffer[16] + ((uint32_t)buffer[17] << 16);
            pF.z = ((uint32_t)buffer[21] << 16);
            pF.w = (uint32_t)buffer[22] + ((uint32_t)buffer[23] << 16);
            y3 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            u2 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
            v2 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
            y4 = (uint32_t)(pF.w & 0x000003FF);

            pF.y = ((uint32_t)buffer[27] << 16);
            y5 = (uint32_t)((pF.y & 0x3FF00000) >> 20);

            // Compute destination indices
            k = tid * 6;
            j = tidd * nDstWidth;
            dpDst[j + k + 0] = lookupTable_cuda[y0];
            dpDst[j + k + 1] = lookupTable_cuda[y1];
            dpDst[j + k + 2] = lookupTable_cuda[y2];
            dpDst[j + k + 3] = lookupTable_cuda[y3];
            dpDst[j + k + 4] = lookupTable_cuda[y4];
            dpDst[j + k + 5] = lookupTable_cuda[y5];

            k = tid * 3;
            j = tidd * nDstWidth / 2;
            dpDst[j + k + 0] = lookupTable_cuda[u0];
            dpDst[j + k + 1] = lookupTable_cuda[u1];
            dpDst[j + k + 2] = lookupTable_cuda[u2];

            j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
            dpDst[j + k + 0] = lookupTable_cuda[v0];
            dpDst[j + k + 1] = lookupTable_cuda[v1];
            dpDst[j + k + 2] = lookupTable_cuda[v2];
        }
    }
    else if (scale == 6) {
        uint32_t v0, y0, u0, y1;
        int nDstH = nDstHeight;
        int nDstW = nDstWidth / 2;
        if (tid < nDstW && tidd < nDstH) {
            int j = tidd * nSrcPitch * scale;
            int k = tid * 16;
            
            // Load source data into shared memory
            buffer[threadIdx.x + 0] = p_Src[j + k + 0];
            buffer[threadIdx.x + 1] = p_Src[j + k + 1];
            buffer[threadIdx.x + 8] = p_Src[j + k + 8];
            buffer[threadIdx.x + 9] = p_Src[j + k + 9];
            
            __syncthreads();

            pF.x = (uint32_t)buffer[0] + ((uint32_t)buffer[1] << 16);
            v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
            y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            u0 = (uint32_t)(pF.x & 0x000003FF);
            pF.x = (uint32_t)buffer[8] + ((uint32_t)buffer[9] << 16);
            y1 = (uint32_t)((pF.x & 0x000FFC00) >> 10);

            // Compute destination indices
            k = tid * 2;
            j = tidd * nDstWidth;
            dpDst[j + k + 0] = lookupTable_cuda[y0];
            dpDst[j + k + 1] = lookupTable_cuda[y1];

            k = tid;
            j = tidd * nDstWidth / 2;
            dpDst[j + k + 0] = lookupTable_cuda[u0];
            j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
            dpDst[j + k + 1] = lookupTable_cuda[v0];
        }
    }
    else if (scale == 2) {
        uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
        int nDstH = nDstHeight;
        int nDstW = nDstWidth / 6;
        if (tid < nDstW && tidd < nDstH) {
            int j = tidd * nSrcPitch * scale;
            int k = tid * 16;

            // Load source data into shared memory
            buffer[threadIdx.x + 0] = p_Src[j + k + 0];
            buffer[threadIdx.x + 1] = p_Src[j + k + 1];
            buffer[threadIdx.x + 3] = p_Src[j + k + 3];
            buffer[threadIdx.x + 5] = p_Src[j + k + 5];
            buffer[threadIdx.x + 6] = p_Src[j + k + 6];
            buffer[threadIdx.x + 7] = p_Src[j + k + 7];
            buffer[threadIdx.x + 8] = p_Src[j + k + 8];
            buffer[threadIdx.x + 9] = p_Src[j + k + 9];
            buffer[threadIdx.x + 10] = p_Src[j + k + 10];
            buffer[threadIdx.x + 11] = p_Src[j + k + 11];
            buffer[threadIdx.x + 12] = p_Src[j + k + 12];
            buffer[threadIdx.x + 14] = p_Src[j + k + 14];
            
            __syncthreads();

            pF.x = (uint32_t)buffer[0] + ((uint32_t)buffer[1] << 16);
            pF.y = ((uint32_t)buffer[3] << 16);
            pF.z = ((uint32_t)buffer[5] << 16);
            pF.w = (uint32_t)buffer[6] + ((uint32_t)buffer[7] << 16);
            v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
            y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            u0 = (uint32_t)(pF.x & 0x000003FF);
            y1 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
            u1 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
            v1 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
            y2 = (uint32_t)(pF.w & 0x000003FF);

            pF.x = (uint32_t)buffer[8] + ((uint32_t)buffer[9] << 16);
            pF.y = (uint32_t)buffer[10] + ((uint32_t)buffer[11] << 16);
            pF.z = (uint32_t)buffer[12];
            pF.w = (uint32_t)buffer[14];
            y3 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            y4 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
            u2 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
            v2 = (uint32_t)(pF.z & 0x000003FF);
            y5 = (uint32_t)(pF.w & 0x000003FF);

            // Compute destination indices
            k = tid * 6;
            j = tidd * nDstWidth;
            dpDst[j + k + 0] = lookupTable_cuda[y0];
            dpDst[j + k + 1] = lookupTable_cuda[y1];
            dpDst[j + k + 2] = lookupTable_cuda[y2];
            dpDst[j + k + 3] = lookupTable_cuda[y3];
            dpDst[j + k + 4] = lookupTable_cuda[y4];
            dpDst[j + k + 5] = lookupTable_cuda[y5];

            k = tid * 3;
            j = tidd * nDstWidth / 2;
            dpDst[j + k + 0] = lookupTable_cuda[u0];
            dpDst[j + k + 1] = lookupTable_cuda[u1];
            dpDst[j + k + 2] = lookupTable_cuda[u2];

            j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
            dpDst[j + k + 0] = lookupTable_cuda[v0];
            dpDst[j + k + 1] = lookupTable_cuda[v1];
            dpDst[j + k + 2] = lookupTable_cuda[v2];
        }
    }
}