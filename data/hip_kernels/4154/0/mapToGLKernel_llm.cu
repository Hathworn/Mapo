#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void mapToGLKernel(uint8_t *dSrc, uint8_t *dDst, int nWidth, int nHeight) {
    // Calculate global thread indices
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check if the thread is within image bounds
    if (tid < nWidth && tidd < nHeight) {
        // Calculate the offset for the source and destination
        int index = (tidd * nWidth + tid) * 3;

        // Copy pixel data
        dDst[index + 0] = dSrc[index + 0];
        dDst[index + 1] = dSrc[index + 1];
        dDst[index + 2] = dSrc[index + 2];
    }
}