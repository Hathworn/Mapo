#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void convertToP208Kernel(uint16_t *pV210, uint16_t *dP208, int nPitch, int nWidth, int nHeight) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;

    // Precompute often used values
    int nDstW = nPitch / 8;
    int nDstH = nHeight;

    // Check valid thread range
    if (tid < nDstW && tidd < nDstH) {
        uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
        uint4 pF;
        
        // Calculate indices only once and reuse
        int k = tid * 8;
        int j = tidd * nPitch;

        // Read 210 pixels packed in 32-bit words
        pF.x = (uint32_t)pV210[j + k + 0] + ((uint32_t)pV210[j + k + 1] << 16);
        pF.y = (uint32_t)pV210[j + k + 2] + ((uint32_t)pV210[j + k + 3] << 16);
        pF.z = (uint32_t)pV210[j + k + 4] + ((uint32_t)pV210[j + k + 5] << 16);
        pF.w = (uint32_t)pV210[j + k + 6] + ((uint32_t)pV210[j + k + 7] << 16);

        // Unpack the values
        v0 = (pF.x >> 20) & 0x3FF;
        y0 = (pF.x >> 10) & 0x3FF;
        u0 = pF.x & 0x3FF;
        y2 = (pF.y >> 20) & 0x3FF;
        u1 = (pF.y >> 10) & 0x3FF;
        y1 = pF.y & 0x3FF;
        u2 = (pF.z >> 20) & 0x3FF;
        y3 = (pF.z >> 10) & 0x3FF;
        v1 = pF.z & 0x3FF;
        y5 = (pF.w >> 20) & 0x3FF;
        v2 = (pF.w >> 10) & 0x3FF;
        y4 = pF.w & 0x3FF;

        // Simplify memory stores using coalesced access patterns
        k = tid * 6;
        j = tidd * (nPitch * 3 / 4);
        dP208[j + k + 0] = y0;
        dP208[j + k + 1] = y1;
        dP208[j + k + 2] = y2;
        dP208[j + k + 3] = y3;
        dP208[j + k + 4] = y4;
        dP208[j + k + 5] = y5;

        k = tid * 3;
        j = tidd * (nPitch * 3 / 8) + nWidth * nHeight;
        dP208[j + k + 0] = u0;
        dP208[j + k + 1] = u1;
        dP208[j + k + 2] = u2;

        j = tidd * (nPitch * 3 / 8) + nWidth * nHeight * 3 / 2;
        dP208[j + k + 0] = v0;
        dP208[j + k + 1] = v1;
        dP208[j + k + 2] = v2;
    }
}