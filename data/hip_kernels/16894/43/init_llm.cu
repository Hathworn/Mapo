#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(uint32_t *v, uint32_t val, uint32_t n) {
    // Compute global thread index
    auto i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single thread to print once
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("init\n");
    }

    // Initialize array elements; avoid branching
    if (i < n) {
        v[i] = val;
    }
}