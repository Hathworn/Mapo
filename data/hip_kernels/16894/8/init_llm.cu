#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(uint32_t *v, uint32_t val, uint32_t n) {
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        v[i] = val;
        // Use early return to optimize instruction flow
        return;
    }
    // Optimize condition checking by reducing unnecessary checks
    if (i == 0) {
        printf("init\n");
    }
}