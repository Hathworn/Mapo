#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(uint32_t *v, uint32_t val, uint32_t n) {
    // Calculate global thread index
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if index is within bounds and initialize value
    if (i < n) {   
        v[i] = val;
    }
    
    // Ensure printf is executed by only one thread
    if (i == 0) {
        printf("init\n");
    }
}