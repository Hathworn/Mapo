#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimized with loop unrolling and strided access
__global__ void square_array(float *a, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;

    // Loop unrolling with strided access
    for (int i = idx; i < N; i += gridSize) {
        a[i] *= a[i];
    }
}