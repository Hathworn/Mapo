#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index more efficiently
    hiprand_init(seed, tidx, 0, &state[tidx]); // Initialize hiprand state with calculated index
}