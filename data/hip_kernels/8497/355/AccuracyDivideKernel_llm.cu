#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Calculate the thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Only first thread performs division
    if (idx == 0) {
        *accuracy /= N;
    }
}