#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
	// Use shared memory to improve memory access performance
        extern __shared__ float sharedSrc[];
        if (idx < srcHeight * srcWidth) {
            sharedSrc[idx] = src[idx];
        }
        __syncthreads();
        tgt[i] = sharedSrc[srcY * srcWidth + srcX];
    }
}