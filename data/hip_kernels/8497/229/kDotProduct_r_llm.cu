#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;
    
    // Optimized loop to populate shared memory
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        sum += a[eidx] * b[eidx];
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Efficient reduction using a single warp
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];

        // Only one thread writes the result
        if (threadIdx.x == 0) {
            target[blockIdx.x] = mysh[0];
        }
    }
}