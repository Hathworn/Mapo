#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate total number of threads
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    // Efficient loop through elements
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Transfer tiled element from source to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}