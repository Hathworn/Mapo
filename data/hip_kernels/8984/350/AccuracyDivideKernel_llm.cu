#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread index to prevent thread divergence and make sure only one thread performs the operation
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}