#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float temp = 0.0f; // Use register for accumulation

    if (eidx < numElements) { 
        // Loop unrolling for improved performance
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) { 
            temp += a[eidx] * b[eidx];
        }
    }
    shmem[threadIdx.x] = temp;
    __syncthreads();

    // Using warp-level reduction for faster execution
    for (uint stride = DP_BLOCKSIZE / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}