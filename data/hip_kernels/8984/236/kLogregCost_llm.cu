#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Optimize: Use boolean and float to reduce branch divergence.
        const bool isMaxLabel = (labelp == maxp);
        int numMax = 0;
        for (int i = 0; i < numOut; i++) {
            numMax += (probs[i * numCases + tx] == maxp);
        }
        correctProbs[tx] = isMaxLabel ? 1.0f / float(numMax) : 0.0f;
    }
}