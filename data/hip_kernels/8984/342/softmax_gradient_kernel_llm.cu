#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    float sum = 0.0f; // Local variable to accumulate the product
    // Unroll loop for better performance
    for (int i = idx; i < dim; i += blockDim.x) {
        sum += dY[i] * Y[i];
    }
    reduction_buffer[idx] = sum;
    __syncthreads();

    // Use warp shuffle for optimized reduction
    if (idx < 32) {
        for (int offset = 32 / 2; offset > 0; offset /= 2) {
            sum += __shfl_down(sum, offset);
        }
    }
    
    if (idx == 0) {
        reduction_buffer[0] = sum;
    }
    __syncthreads();

    // Compute gradient
    float tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}