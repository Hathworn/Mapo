#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matSum(float* S, float* A, float* B, int N) {
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate the linear thread id based on 2D grid for N x N matrix
    if (i < N && j < N) { // Add boundary check to prevent out-of-bounds
        int tid = i * N + j; // Use tid calculation within boundary check
        S[tid] = A[tid] + B[tid];
    }
}