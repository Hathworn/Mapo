#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        // Load data into registers for faster access
        float a = A[i];
        float b = B[i];
        C[i] = a + b;
    }
}