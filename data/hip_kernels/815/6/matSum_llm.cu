#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matSum(float* S, float* A, float* B, int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Simplify calculation of global thread index
    if (i < N && j < N) {
        int tid = i * N + j;
        S[tid] = A[tid] + B[tid];
    }
}