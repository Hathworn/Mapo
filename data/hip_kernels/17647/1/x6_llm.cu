#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void x6(float* x7, float* x8, float* x9, int x10) {
    int x12 = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use stride loop for better performance and coalesced memory access.
    for (int stride = gridDim.x * blockDim.x; x12 < x10; x12 += stride) {
        x9[x12] = x7[x12] + x8[x12];
    }
}