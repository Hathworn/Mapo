#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void x2(float* x3, float x4, int x5) {
    // Calculate the global thread index
    int x7 = blockIdx.x * blockDim.x + threadIdx.x;
    // Loop over each stride within the grid
    for (; x7 < x5; x7 += blockDim.x * gridDim.x) {
        x3[x7] = x4;
    }
}