#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void x24(float* x25, float* x26, float* x27, int x28) {
    int x30 = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    // Use a stride loop for better memory access pattern
    for (int i = x30; i < x28; i += stride) {
        x27[i] = x25[i] * x26[i];
    }
}