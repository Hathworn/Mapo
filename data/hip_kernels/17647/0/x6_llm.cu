#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**************** Snippet ****************/
__global__ void x6(float* x7, float* x8, float* x9, int x10) {
    // Calculate global index for thread
    int x12 = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x; // Pre-compute stride

    // Use stride loop for processing
    for (int i = x12; i < x10; i += stride) {
        x9[i] = x7[i] + x8[i];
    }
}