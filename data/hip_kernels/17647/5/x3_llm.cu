#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void x3(int* x4, int x5, int x6) {
    int x7 = gridDim.x * blockDim.x;
    int x8 = threadIdx.x + blockIdx.x * blockDim.x;
    int x9 = -x5;
    // Unroll the loop to improve performance
    #pragma unroll
    while (x8 < x6) {
        int x10 = x8;
        int xVal = x4[x10];
        // Combine comparisons and assignments to optimize
        x4[x10] = (xVal > x5) ? x5 : ((xVal < x9) ? x9 : xVal);
        x8 += x7;
    }
}