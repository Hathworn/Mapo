#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void x15(float* x16, float* x17, float* x18, int x19) {
    // Calculate the global thread index once
    int x21 = threadIdx.x + blockIdx.x * blockDim.x;

    // Use strided loop for all thread calculations in parallel
    for (int x22 = x21; x22 < x19; x22 += gridDim.x * blockDim.x) {
        // Perform subtraction and store the result
        x18[x22] = x16[x22] - x17[x22];
    }
}