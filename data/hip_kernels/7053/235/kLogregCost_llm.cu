#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Improved branch by setting default and only checking conditionally
        int numMax = (labelp == maxp);
        float correctProb = 0.0f;

        // Combined loop to avoid conditional execution overhead
        for (int i = 0; i < numOut; i++) {
            numMax += (probs[i * numCases + tx] == maxp);
        }
        
        if (labelp == maxp) {
            correctProb = 1.0f / float(numMax);
        }

        correctProbs[tx] = correctProb;
    }
}