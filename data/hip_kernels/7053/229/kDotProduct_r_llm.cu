#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float accum = 0.0f;

    // Loop over elements with stride, accumulate in register
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        accum += a[eidx] * b[eidx];
    }
    shmem[threadIdx.x] = accum;
    __syncthreads();

    // Unrolling reduction with warp efficiency
    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    __syncthreads();

    if (threadIdx.x < 32) {
        volatile float* mysh = shmem; // Use volatile pointer for warp reduction
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}