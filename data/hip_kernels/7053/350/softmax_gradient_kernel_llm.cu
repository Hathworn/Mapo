#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    extern __shared__ float reduction_buffer[]; // Use dynamic shared memory
    const int idx = threadIdx.x;
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    float tmp = 0;

    // Unroll loop for better performance; vectorize memory access
    for (int i = idx; i < dim; i += blockDim.x * 4) {
        float local_tmp = 0;
        if (i < dim) local_tmp += dY[i] * Y[i];
        if (i + blockDim.x < dim) local_tmp += dY[i + blockDim.x] * Y[i + blockDim.x];
        if (i + 2 * blockDim.x < dim) local_tmp += dY[i + 2 * blockDim.x] * Y[i + 2 * blockDim.x];
        if (i + 3 * blockDim.x < dim) local_tmp += dY[i + 3 * blockDim.x] * Y[i + 3 * blockDim.x];
        tmp += local_tmp;
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Use warp-level reductions
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }

    // Compute the gradient and store the result
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}