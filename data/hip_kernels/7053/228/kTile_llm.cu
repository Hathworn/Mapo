#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate the global index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total number of threads
    const int numThreads = blockDim.x * gridDim.x;
    // Precompute the total number of elements to reduce computations inside the loop
    const uint numEls = tgtWidth * tgtHeight;

    for (uint i = idx; i < numEls; i += numThreads) {
        // Use division and modulus to determine x and y coordinates
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        // Utilize modulo operations to handle repetition over source dimensions
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Assign the corresponding value from the source to the target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}