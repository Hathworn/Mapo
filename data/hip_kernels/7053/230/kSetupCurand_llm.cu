#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    
    // Optimize by ensuring this kernel only runs for active threads
    if (tidx < TOTAL_NUM_RND_THREADS) {
        hiprand_init(seed, tidx, 0, &state[tidx]);
    }
}