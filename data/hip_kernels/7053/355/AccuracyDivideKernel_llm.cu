#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Check if the thread is the first one
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        // Perform division in the first thread of the first block
        *accuracy /= N;
    }
}