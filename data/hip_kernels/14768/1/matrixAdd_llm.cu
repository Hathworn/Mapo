#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixAdd(int *a, int *b, int *c, int N)
{
    // Calculate global thread ID
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread is within matrix bounds
    if (col < N && row < N)
    {
        int index = col + row * N; // Compute linear index for 2D grid
        c[index] = a[index] + b[index]; // Perform addition
    }
}