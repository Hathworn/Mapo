#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void quickSort(int *x, int *dfirst, int *dlast, int *list)
{
    // Get thread index
    int idx = threadIdx.x;
    int first = dfirst[idx];
    int last = dlast[idx];
    list[idx] = 0;

    // Perform sorting if first index is less than last
    if (first < last)
    {
        // Initializations
        int pivot = first;
        int i = first;
        int j = last;
        int temp;

        // Repeatedly partition array until completed
        while (i < j)
        {
            while (x[i] <= x[pivot] && i < last)
                i++; // Increment i while condition holds

            while (x[j] > x[pivot])
                j--; // Decrement j while condition holds

            // Swap elements at i and j
            if (i < j)
            {
                temp = x[i];
                x[i] = x[j];
                x[j] = temp;
            }
        }

        // Swap pivot with element at j
        temp = x[pivot];
        x[pivot] = x[j];
        x[j] = temp;

        // Determine if further sorting is needed
        for (i = first; i <= last - 1; i++) // Use last - 1 for index correctness
        {
            if (x[i] > x[i + 1])
            {
                list[idx] = j + 1;
                break; // Break loop once condition is met
            }
        }
    }
}