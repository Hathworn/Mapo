#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void enumsort(int *deva, int *devn)
{
    int tid = threadIdx.x;
    int i, count = 0;

    // Use shared memory for better performance
    extern __shared__ int shared_deva[];

    // Load data into shared memory
    shared_deva[tid] = deva[tid];
    __syncthreads();

    // Perform the counting sort
    for (i = 0; i < N; i++) {
        if ((shared_deva[i] <= shared_deva[tid]) && (i != tid)) {
            count++;
        }
    }

    // Write the sorted value to devn
    devn[count] = shared_deva[tid];
}