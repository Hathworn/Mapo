#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void projectionProfileVerticalCuda(const uint8_t * image, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * projection)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Use shared memory for efficient atomic operations
    __shared__ uint32_t sharedProjection[1024]; // Adjust size according to maximum number of threads per block
    if (y < height) {
        sharedProjection[threadIdx.y] = 0; // Initialize shared projection for each thread within the block
    }
    __syncthreads(); // Synchronize to ensure initialization is complete
    
    if (x < width && y < height) {
        const uint8_t * imageY = image + y * rowSize + x;
        atomicAdd(&sharedProjection[threadIdx.y], (*imageY)); // Accumulate in shared memory
    }
    __syncthreads(); // Synchronize to ensure all operations are complete before writing to global memory
    
    if (y < height) {
        atomicAdd(&projection[y], sharedProjection[threadIdx.y]); // Commit to global memory
    }
}