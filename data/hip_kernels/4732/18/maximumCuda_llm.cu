#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void maximumCuda(const uint8_t* in1, uint32_t rowSizeIn1, const uint8_t* in2, uint32_t rowSizeIn2, uint8_t* out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate global thread index
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds to ensure threads operate within image dimensions
    if (x < width && y < height) {
        // Compute offset only once
        uint32_t offsetIn1 = y * rowSizeIn1 + x;
        uint32_t offsetIn2 = y * rowSizeIn2 + x;
        uint32_t offsetOut = y * rowSizeOut + x;

        // Assign maximum value to output
        out[offsetOut] = max(in1[offsetIn1], in2[offsetIn2]);
    }
}