#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void minimumCuda(const uint8_t* __restrict__ in1, uint32_t rowSizeIn1, const uint8_t* __restrict__ in2, uint32_t rowSizeIn2, uint8_t* __restrict__ out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure thread is within image bounds
    if (x < width && y < height) {
        // Use DOF (pointer differential offset) and reduce pointer dereference
        uint32_t offset1 = y * rowSizeIn1 + x;
        uint32_t offset2 = y * rowSizeIn2 + x;
        uint32_t offsetOut = y * rowSizeOut + x;
        
        out[offsetOut] = min(in1[offset1], in2[offset2]); // Utilize min intrinsic
    }
}