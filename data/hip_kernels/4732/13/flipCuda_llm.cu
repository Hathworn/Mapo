#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void flipCuda(const uint8_t* __restrict__ in, uint32_t rowSizeIn, uint8_t* __restrict__ out, uint32_t rowSizeOut, uint32_t width, uint32_t height, bool horizontal, bool vertical)
{
    // Calculate input coordinates
    const uint32_t inX = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t inY = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure threads operate only within bounds
    if (inX < width && inY < height) {
        // Compute corresponding output coordinates using conditional operators
        const uint32_t outX = horizontal ? (width - 1 - inX) : inX;
        const uint32_t outY = vertical   ? (height - 1 - inY) : inY;

        // Perform the pixel flipping
        out[outY * rowSizeOut + outX] = in[inY * rowSizeIn + inX];
    }
}