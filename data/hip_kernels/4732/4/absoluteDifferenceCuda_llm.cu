#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void absoluteDifferenceCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        // Prefetch memory addresses
        const uint32_t indexIn1 = y * rowSizeIn1 + x;
        const uint32_t indexIn2 = y * rowSizeIn2 + x;
        const uint32_t indexOut = y * rowSizeOut + x;
        
        // Compute absolute difference using prefetch indices
        out[indexOut] = (in1[indexIn1] > in2[indexIn2]) ? 
                        (in1[indexIn1] - in2[indexIn2]) : 
                        (in2[indexIn2] - in1[indexIn1]);
    }
}