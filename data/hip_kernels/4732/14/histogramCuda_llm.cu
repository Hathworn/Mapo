#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogramCuda(const uint8_t * data, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * histogram)
{
    // Calculate global thread index for 1D data
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure within bounds before processing
    if (x < width && y < height) {
        const uint32_t id = y * rowSize + x;
        // Use atomic operations for safely updating histogram in parallel
        atomicAdd(&histogram[data[id]], 1);
    }
}