#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setPixelCuda(uint8_t *in, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t *pointX, uint32_t *pointY, uint32_t pointSize, uint32_t value)
{
    // Calculate unique global index for each thread
    uint32_t idPoint = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread processes only valid data
    if (idPoint < pointSize) 
    {
        uint32_t x = pointX[idPoint];
        uint32_t y = pointY[idPoint];

        // Validate coordinates are within bounds
        if (x < width && y < height)
        {
            // Access memory efficiently
            in[y * rowSize + x] = value;
        }
    }
}