#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void isAnyEqualCuda(const uint8_t *image, const uint8_t *value, size_t valueCount, uint32_t width, uint32_t height, uint32_t *differenceCount)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Launch bounds check
    if (x < width && y < height) {
        const uint32_t id = y * width + x;

        uint8_t pixelValue = image[id];
        bool equal = false;

        // Use shared memory to minimize global memory access
        extern __shared__ uint8_t sharedValue[];
        uint32_t tid = threadIdx.y * blockDim.x + threadIdx.x;
        for (uint32_t i = tid; i < valueCount; i += blockDim.x * blockDim.y) {
            sharedValue[i] = value[i];
        }
        __syncthreads();
        
        // Unroll loop for better performance
        for (uint32_t i = 0; i < valueCount; ++i) {
            if (pixelValue == sharedValue[i]) {
                equal = true;
                break;
            }
        }

        // Atomic operation if condition is met
        if (equal) {
            atomicAdd(differenceCount, 1);
        }
    }
}