#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void isEqualCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint32_t width, uint32_t height, uint32_t * isEqual )
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Use shared memory for intermediate results to reduce atomic operations
    __shared__ uint32_t sharedIsEqual;

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        sharedIsEqual = 1; // Initialize shared memory
    }
    __syncthreads();

    if ( x < width && y < height ) {
        // Check equality and reduce directly in shared memory
        if(in1[y * rowSizeIn1 + x] != in2[y * rowSizeIn2 + x]) {
            atomicAnd(&sharedIsEqual, 0);
        }
    }

    __syncthreads();

    // Use block-level reduction to minimize global memory atomic operations
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        atomicAnd(isEqual, sharedIsEqual);
    }
}