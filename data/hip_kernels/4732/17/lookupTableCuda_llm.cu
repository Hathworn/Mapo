#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lookupTableCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height, uint8_t *table)
{
    // Calculate unique thread index
    const uint32_t idx = blockIdx.y * blockDim.y * rowSizeIn + blockIdx.x * blockDim.x;
    const uint32_t tid = threadIdx.y * rowSizeIn + threadIdx.x;

    const uint32_t index = idx + tid;

    // Check bounds
    if (index < width * height) {
        out[index] = table[in[index]];
    }
}