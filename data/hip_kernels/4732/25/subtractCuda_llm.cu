#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtractCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate global thread index
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure the thread is within bounds
    if (x < width && y < height) {
        // Directly calculate the index once
        uint32_t indexIn1 = y * rowSizeIn1 + x;
        uint32_t indexIn2 = y * rowSizeIn2 + x;
        uint32_t indexOut = y * rowSizeOut + x;

        // Perform the subtraction with a ternary operation
        out[indexOut] = (in1[indexIn1] > in2[indexIn2]) ? (in1[indexIn1] - in2[indexIn2]) : 0;
    }
}