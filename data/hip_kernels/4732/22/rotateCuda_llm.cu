#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rotateCuda(const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, float inXStart, float inYStart, uint32_t width, uint32_t height, float cosAngle, float sinAngle)
{
    uint32_t outX = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t outY = blockDim.y * blockIdx.y + threadIdx.y;

    // Only process valid pixels in the output
    if (outX < width && outY < height) {
        // Calculate input coordinates using inverse rotation
        float exactInX = inXStart + cosAngle * outX + sinAngle * outY;
        float exactInY = inYStart - sinAngle * outX + cosAngle * outY;

        int32_t inX = static_cast<int32_t>(exactInX);
        int32_t inY = static_cast<int32_t>(exactInY);

        // Calculate output offset
        uint8_t* outPtr = out + outY * rowSizeOut + outX;

        // Check input boundaries for bilinear interpolation
        if (inX < 0 || inX >= width - 1 || inY < 0 || inY >= height - 1) {
            *outPtr = 0; // Set value to 0 if out of bounds
        } else {
            // Calculate input offset
            const uint8_t* inPtr = in + inY * rowSizeIn + inX;

            // Bilinear interpolation
            float probX = exactInX - inX;
            float probY = exactInY - inY;
            float mean =  *inPtr * (1 - probX) * (1 - probY) +
                         *(inPtr + 1) * probX * (1 - probY) +
                         *(inPtr + rowSizeIn) * (1 - probX) * probY +
                         *(inPtr + rowSizeIn + 1) * probX * probY + 0.5f;

            *outPtr = static_cast<uint8_t>(mean);
        }
    }
}