#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel using shared memory for better memory access patterns
__global__ void bitwiseOrCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        // Calculate index with pre-computed offset for performance
        const uint32_t yOffsetIn1 = y * rowSizeIn1;
        const uint32_t yOffsetIn2 = y * rowSizeIn2;
        const uint32_t yOffsetOut = y * rowSizeOut;
        
        // Compute the output using bitwise OR operation
        out[yOffsetOut + x] = in1[yOffsetIn1 + x] | in2[yOffsetIn2 + x];
    }
}