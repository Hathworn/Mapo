#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitwiseAndCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate unique thread index
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Check within bounds
    if (x < width && y < height) {
        // Compute index once and reuse
        const uint32_t yOffsetIn1 = y * rowSizeIn1;
        const uint32_t yOffsetIn2 = y * rowSizeIn2;
        const uint32_t yOffsetOut = y * rowSizeOut;
        const uint32_t idIn1 = yOffsetIn1 + x;
        const uint32_t idIn2 = yOffsetIn2 + x;
        const uint32_t idOut = yOffsetOut + x;

        // Perform bitwise and operation
        out[idOut] = in1[idIn1] & in2[idIn2];
    }
}