#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void projectionProfileHorizontalCuda(const uint8_t * image, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * projection)
{
    // Calculate unique thread index for x
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;

    // Use a single loop to iterate over y-coordinate
    if (x < width) {
        uint32_t sum = 0;
        for (uint32_t y = 0; y < height; ++y) {
            const uint8_t * imageX = image + y * rowSize + x;
            sum += *imageX;
        }
        atomicAdd(&projection[x], sum);
    }
}