#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void invertCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate flattened index for 1D memory access
    const uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;

    // Check bounds for valid pixel processing
    if (idx < width && idy < height) {
        // Compute linear memory coordinate and perform inversion
        const uint32_t indexIn = idy * rowSizeIn + idx;
        const uint32_t indexOut = idy * rowSizeOut + idx;
        out[indexOut] = ~in[indexIn];
    }
}