#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyCuda(const uint8_t* __restrict__ in, uint32_t rowSizeIn, uint8_t* __restrict__ out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Early exit to improve efficiency
    if (x >= width || y >= height) return;

    // Optimize memory access using shared memory
    __shared__ uint8_t sharedMem[BLOCK_SIZE][BLOCK_SIZE];

    // Load data to shared memory
    sharedMem[threadIdx.y][threadIdx.x] = in[y * rowSizeIn + x];
    __syncthreads();

    // Write data from shared memory
    out[y * rowSizeOut + x] = sharedMem[threadIdx.y][threadIdx.x];
}