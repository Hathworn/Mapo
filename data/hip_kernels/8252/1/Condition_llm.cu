#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef TIME
#define COMM 1
#elif NOTIME
#define COMM 0
#endif

#define MASK_WIDTH 5
#define TILE_WIDTH 32
#define GPU 1
#define COMMENT "skeletization_GPU"
#define RGB_COMPONENT_COLOR 255

typedef struct {
    unsigned char red, green, blue;
} PPMPixel;

typedef struct {
    int x, y;
    PPMPixel *data;
} PPMImage;

typedef struct {
    int x, y;
} Par;

double time_total;

// Optimized global kernel function
__global__ void Condition(int *GrayScale_, int *d_changing1, int *cont, int linhas, int colunas, int flag)
{
    // Pre-calculated offsets to access neighboring pixels
    constexpr int X_index[] = {-1, -1, 0, 1, 1, 1, 0, -1};
    constexpr int Y_index[] = {0, 1, 1, 1, 0, -1, -1, -1};

    // Use shared memory for neighbor values to reduce redundant global memory accesses
    __shared__ int neighbors_shared[TILE_WIDTH][TILE_WIDTH][8];
    
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    int fil = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int index = fil * colunas + col;

    if (fil > 0 && col > 0 && fil < linhas-1 && col < colunas-1)
    {
        d_changing1[index] = 0;
        
        // Load neighbors into shared memory
        for (int i = 0; i < 8; i++) {
            neighbors_shared[threadIdx.y][threadIdx.x][i] = GrayScale_[(fil + X_index[i]) * colunas + (col + Y_index[i])];
        }
        __syncthreads();

        // Calculate total and ans using shared memory
        int total = 0, ans = 0;
        for(int i = 0; i < 8; i++) {
            total += neighbors_shared[threadIdx.y][threadIdx.x][i];
            if (i < 7 && neighbors_shared[threadIdx.y][threadIdx.x][i] == 0 && neighbors_shared[threadIdx.y][threadIdx.x][i+1] == 1) {
                ans++;
            }
        }
        if(neighbors_shared[threadIdx.y][threadIdx.x][7] == 0 && neighbors_shared[threadIdx.y][threadIdx.x][0] == 1) {
            ans++;
        }

        if(flag != 1) {
            if(GrayScale_[fil * colunas + col] == 1 && total >= 2 && total <= 6 && ans == 1 &&
               neighbors_shared[threadIdx.y][threadIdx.x][0] * neighbors_shared[threadIdx.y][threadIdx.x][2] * neighbors_shared[threadIdx.y][threadIdx.x][4] == 0 &&
               neighbors_shared[threadIdx.y][threadIdx.x][2] * neighbors_shared[threadIdx.y][threadIdx.x][4] * neighbors_shared[threadIdx.y][threadIdx.x][6] == 0) {
                d_changing1[index] = 1;
                cont[flag] = 1;
            }
        }
        else {
            if(GrayScale_[fil * colunas + col] == 1 && total >= 2 && total <= 6 && ans == 1 &&
               neighbors_shared[threadIdx.y][threadIdx.x][0] * neighbors_shared[threadIdx.y][threadIdx.x][2] * neighbors_shared[threadIdx.y][threadIdx.x][6] == 0 &&
               neighbors_shared[threadIdx.y][threadIdx.x][0] * neighbors_shared[threadIdx.y][threadIdx.x][4] * neighbors_shared[threadIdx.y][threadIdx.x][6] == 0) {
                d_changing1[index] = 1;
                cont[flag] = 1;
            }
        }
    }
}