#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VectorAdd(int *a, int *b, int *c, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x; // compute global index
    if (i < n)
        c[i] = a[i] + b[i]; // perform vector addition
}