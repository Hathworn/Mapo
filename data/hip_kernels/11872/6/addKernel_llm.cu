#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global index for this thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform vector addition
    if (i < N) // Ensure within array bounds assuming N is defined
    {
        c[i] = a[i] + b[i];
    }
}