#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimized with grid-stride loop
__global__ void VectorAdd(int *a, int *b, int *c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Using grid-stride loop for better scalability
    for (; i < n; i += stride) {
        c[i] = a[i] + b[i];
    }
}