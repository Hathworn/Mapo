#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Compute global thread index to handle larger arrays
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate total number of threads
    int gridSize = blockDim.x * gridDim.x;

    // Loop to cover all elements (useful for arrays larger than the number of threads)
    while (i < N) {  // Assume N is defined elsewhere
        c[i] = a[i] + b[i];
        i += gridSize;
    }
}