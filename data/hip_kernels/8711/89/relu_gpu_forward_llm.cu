#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Using shared memory to reduce global memory access
__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    extern __shared__ float shared_in[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    if (tid < N) {
        // Load input data into shared memory
        shared_in[local_tid] = in[tid];
        __syncthreads(); // Ensure all threads have loaded data

        // Compute ReLU using shared memory
        out[tid] = shared_in[local_tid] > 0 ? shared_in[local_tid] : 0;
    }
}