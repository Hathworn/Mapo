#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    // Cache the thread ID for reuse
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Streamline condition check and assignment
    if (tid < N) {
        float data = indata[tid];
        ingrad[tid] = (data > 0) ? outgrad[tid] : 0;
    }
}