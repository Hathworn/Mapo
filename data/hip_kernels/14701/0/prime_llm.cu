#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

void printUsage(char* appName);
int parseArgs(char** argv,int* pSize,int *print,int argc);
uint64_t getTime();

__global__ void prime(int *a, int count)
{
    // Calculate the global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Return if the thread index is out of bounds
    if (tid >= count) return;

    // Cache the candidate number
    int can = a[tid];

    // Quick elimination for even numbers
    if (can > 2 && can % 2 == 0) 
    {
        a[tid] = 1;
        return;
    }

    // Initialize counter and calculate limit
    int counter = 3;
    float limit = sqrtf(can) + 1;

    // Loop through odd numbers only
    for (; counter <= limit; counter += 2)
    {
        if (can % counter == 0)
        {
            a[tid] = 1; // Set as non-prime
            return;
        }
    }
}