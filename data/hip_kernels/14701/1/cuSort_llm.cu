#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuSort(float* data, int bucketSize, int* startPoint) 
{
    // Use shared memory for more efficient data access
    extern __shared__ float sharedData[];
    int L = blockIdx.x * bucketSize;
    int U = L + bucketSize;
    int tid = threadIdx.x;
    float tmp;
    startPoint[blockIdx.x] = L;

    // Load data into shared memory
    if (L + tid < U) {
        sharedData[tid] = data[L + tid];
    }
    __syncthreads();

    // Sort using insertion sort in shared memory
    for (int i = 1; i < bucketSize; i++) {
        tmp = sharedData[i];
        int j = i - 1;
        while (j >= 0 && sharedData[j] > tmp) {
            sharedData[j + 1] = sharedData[j];
            j--;
        }
        sharedData[j + 1] = tmp;
    }
    __syncthreads();

    // Write sorted data back to global memory
    if (L + tid < U) {
        data[L + tid] = sharedData[tid];
    }
}