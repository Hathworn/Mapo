#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Use thread ID to allow vectorized operations for larger arrays
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform addition on each element assuming arrays not scalars
    c[idx] = a[idx] + b[idx];
}