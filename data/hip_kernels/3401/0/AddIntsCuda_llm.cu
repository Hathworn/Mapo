#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void AddIntsCuda(int *a, int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if (i < N) // Ensure within bounds, N should be defined as the size of the arrays
    {
        a[i] += b[i];
    }
}