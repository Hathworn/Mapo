#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(int order, const int matrices, double * C)
{
    // Calculate the unique thread index for device memory operations
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIdy = blockIdx.y * blockDim.y + threadIdx.y;
    int stride = blockDim.x * gridDim.x; // Stride for memory access optimization

    // Iterate over matrix batch efficiently using stride
    for (int b = 0; b < matrices; ++b) {
        // Check combined row and column boundary condition
        int index = b * order * order + globalIdx * order + globalIdy;
        if ((globalIdx < order) && (globalIdy < order)) {
            C[index] = 0;
        }
        globalIdx += stride;  // Efficiently skip over threads out-of-bounds
    }
}