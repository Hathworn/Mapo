#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_Heat (float *u, float *utmp, float *residual, int N) {
    // Calculate global row and column indices
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if indices are within valid range
    if (i > 0 && i < N-1 && j > 0 && j < N-1) {
        // Calculate linear index for current element
        int idx = i * N + j;

        // Compute updated value using stencil
        utmp[idx] = 0.25f * (u[idx - 1] +  // left
                             u[idx + 1] +  // right
                             u[idx - N] +  // top
                             u[idx + N]);  // bottom

        // Calculate difference and update residual
        float diff = utmp[idx] - u[idx];
        residual[idx] = diff * diff;
    }
}