#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_HeatReduction(float *res, float *result) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input into shared memory
    sdata[tid] = res[index];
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        result[blockIdx.x] = sdata[0];
    }
}