#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // calculate global thread ID
    if (tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
}