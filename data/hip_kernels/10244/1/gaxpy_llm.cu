#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpy(double *y, double *a, double *x, int m, int n) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];

    // Early exit if block index exceeds the limit
    if (bid >= m) return;
    
    // Perform calculations only on valid thread indices
    if (tid < n) {
        dots_s[tid] = a[bid * n + tid] * x[tid];
    }
    __syncthreads();

    // Use reduction to sum the shared memory elements
    for (int stride = n / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            dots_s[tid] += dots_s[tid + stride];
        }
        __syncthreads();
    }

    // Write the result from tid 0 to global memory
    if (tid == 0) {
        y[bid] = dots_s[0];
    }
}