#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void suma(int a, int b, int *c) {
    // Calculate the linear index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Only thread 0 writes the result, preventing unnecessary threads
    if (idx == 0) {
        *c = a + b;
    }
}