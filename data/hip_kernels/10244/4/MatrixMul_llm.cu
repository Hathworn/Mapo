#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMul(float *Md, float *Nd, float *Pd, const int WIDTH) {
    // Calculate thread id
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

    // Initialize Pvalue to accumulate the result
    float Pvalue = 0;

    // Use shared memory for tiling
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    for (int m = 0; m < WIDTH / TILE_WIDTH; ++m) {
        // Collaboratively load tiles into shared memory
        if (row < WIDTH && m * TILE_WIDTH + threadIdx.x < WIDTH)
            Mds[threadIdx.y][threadIdx.x] = Md[row * WIDTH + m * TILE_WIDTH + threadIdx.x];
        else
            Mds[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < WIDTH && m * TILE_WIDTH + threadIdx.y < WIDTH)
            Nds[threadIdx.y][threadIdx.x] = Nd[(m * TILE_WIDTH + threadIdx.y) * WIDTH + col];
        else
            Nds[threadIdx.y][threadIdx.x] = 0.0f;

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the tiles
        for (int k = 0; k < TILE_WIDTH; ++k) {
            Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
        }

        // Synchronize to ensure that the preceding computation is done before loading two new tiles
        __syncthreads();
    }

    // Write the block's result to device memory
    if (row < WIDTH && col < WIDTH)
        Pd[row * WIDTH + col] = Pvalue;
}