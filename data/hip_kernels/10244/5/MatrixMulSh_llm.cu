#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Define TILE_WIDTH for the shared memory tiles
#define TILE_WIDTH 16

__global__ void MatrixMulSh( float *Md, float *Nd, float *Pd, const int WIDTH )
{
    // Shared memory tiles for sub-matrices
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // Calculate thread indices for matrix
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

    // Initialize element accumulator
    float Pvalue = 0.0f;

    for (int m = 0; m < WIDTH / TILE_WIDTH; m++)
    {
        // Load Md and Nd matrices into shared memory
        Mds[threadIdx.y][threadIdx.x] = Md[row * WIDTH + (m * TILE_WIDTH + threadIdx.x)];
        Nds[threadIdx.y][threadIdx.x] = Nd[(m * TILE_WIDTH + threadIdx.y) * WIDTH + col];
        __syncthreads(); // Synchronize threads

        // Compute sub-matrix multiplication
        for (int k = 0; k < TILE_WIDTH; k++)
        {
            Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
        }
        __syncthreads(); // Synchronize threads
    }

    // Write result to Pd matrix
    Pd[row * WIDTH + col] = Pvalue;
}