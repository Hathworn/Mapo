#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_print( long const* p, int n)
{
    int idx = threadIdx.x; // Get thread index
    if (idx < n) // Check if index is within bounds
    {
        printf("Thread %d: %ld\n", idx, p[idx]); // Print thread index and value
    }
    if (idx == 0) // Only one thread prints the number of longs
    {
        printf("long count: %d\n", n); // Print number of longs
    }
}