#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_print(int const* p, int n)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global index for parallel print

    if (idx == 0) 
    {
        printf("int: %d ", n); // Only the first thread prints the integer
    }

    // Synchronize before printing array
    __syncthreads(); 

    if (idx < n) 
    {
        printf("%d ", p[idx]); // Each thread prints one element
    }
}