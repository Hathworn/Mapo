#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_print(size_t const* p, int n)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId < n)
    {
        printf("Index %d: %lu\n", threadId, p[threadId]);  // Improved memory access using index
    }
}