#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel to compute out_lod0
__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod, const size_t lod_len, size_t* out_lod0) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure thread index is within bounds
    if (index < lod_len) {
        // Preload in_lod value for better memory efficiency
        size_t in_value = in_lod[index];
        out_lod0[index] = in_value - num_erased[in_value];
    }
}