#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if (i < N) // Ensure index is within bounds
    {
        c[i] = a[i] + b[i];
    }
}