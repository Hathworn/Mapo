#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

__device__ void swap(int *a, int *b) {
    int temp = *a;
    *a = *b;
    *b = temp;
}

__global__ void sort(int *d_arr, int n, bool isEven) {
    int stride = blockDim.x * 2;
    int start = isEven ? threadIdx.x * 2 : threadIdx.x * 2 + 1;
    // Loop through all elements in the array with step size of stride
    for (int i = start; i < n - 1; i += stride) {
        // Swap elements to sort
        if (d_arr[i] > d_arr[i + 1]) {
            swap(&d_arr[i], &d_arr[i + 1]);
        }
    }
}