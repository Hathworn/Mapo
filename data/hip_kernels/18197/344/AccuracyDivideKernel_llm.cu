#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use a single thread to safely perform division
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}