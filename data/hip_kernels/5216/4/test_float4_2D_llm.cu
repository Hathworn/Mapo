#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCuda - Java bindings for NVIDIA CUDA driver and runtime API
* http://www.jcuda.org
*
* Copyright 2010 Marco Hutter - http://www.jcuda.org
*/

/**
* Kernels for the JCudaDriverTextureTest class. These
* kernels will read data via the texture references at
* the given positions, and store the value that is
* read into the given output memory.
*/

texture<float,  1, hipReadModeElementType> texture_float_1D;
texture<float,  2, hipReadModeElementType> texture_float_2D;
texture<float,  3, hipReadModeElementType> texture_float_3D;

texture<float4, 1, hipReadModeElementType> texture_float4_1D;
texture<float4, 2, hipReadModeElementType> texture_float4_2D;
texture<float4, 3, hipReadModeElementType> texture_float4_3D;

// Optimize the kernel to compute and store output based on block and thread indices
__global__ void test_float4_2D(float4 *output, float posX, float posY)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    float4 result = tex2D(texture_float4_2D, posX + idx, posY);  // Sample texture with offset based on index
    output[idx] = result;  // Store the calculated result
}