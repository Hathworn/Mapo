#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCuda - Java bindings for NVIDIA CUDA driver and runtime API
* http://www.jcuda.org
*
* Copyright 2010 Marco Hutter - http://www.jcuda.org
*/

/**
* Kernels for the JCudaDriverTextureTest class. These
* kernels will read data via the texture references at
* the given positions, and store the value that is
* read into the given output memory.
*/

texture<float,  1, hipReadModeElementType> texture_float_1D;
texture<float,  2, hipReadModeElementType> texture_float_2D;
texture<float,  3, hipReadModeElementType> texture_float_3D;

texture<float4, 1, hipReadModeElementType> texture_float4_1D;
texture<float4, 2, hipReadModeElementType> texture_float4_2D;
texture<float4, 3, hipReadModeElementType> texture_float4_3D;

extern "C"

__global__ void test_float_2D(float *output, float posX, float posY)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Fetch result only for the first thread
    if (idx == 0) {
        float result = tex2D(texture_float_2D, posX, posY);
        output[0] = result;
    }
}