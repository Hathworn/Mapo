#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

texture<float,  1, hipReadModeElementType> texture_float_1D;
texture<float,  2, hipReadModeElementType> texture_float_2D;
texture<float,  3, hipReadModeElementType> texture_float_3D;

texture<float4, 1, hipReadModeElementType> texture_float4_1D;
texture<float4, 2, hipReadModeElementType> texture_float4_2D;
texture<float4, 3, hipReadModeElementType> texture_float4_3D;

extern "C"

__global__ void test_float4_1D(float4 *output, float posX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global index
    if (idx == 0) // Ensure only one thread writes to output
    {
        float4 result = tex1D(texture_float4_1D, posX); // Read from texture
        output[0] = result; // Store result in output
    }
}