#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* JCuda - Java bindings for NVIDIA CUDA driver and runtime API
* http://www.jcuda.org
*
* Copyright 2010 Marco Hutter - http://www.jcuda.org
*/

/**
* Kernels for the JCudaDriverTextureTest class. These
* kernels will read data via the texture references at
* the given positions, and store the value that is
* read into the given output memory.
*/

texture<float,  1, hipReadModeElementType> texture_float_1D;
texture<float,  2, hipReadModeElementType> texture_float_2D;
texture<float,  3, hipReadModeElementType> texture_float_3D;

texture<float4, 1, hipReadModeElementType> texture_float4_1D;
texture<float4, 2, hipReadModeElementType> texture_float4_2D;
texture<float4, 3, hipReadModeElementType> texture_float4_3D;

extern "C"

__global__ void test_float4_3D(float4 *output, float posX, float posY, float posZ)
{
    // Calculate the index for the output using unique thread identifiers
    const int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Read texture at the given position
    float4 result = tex3D(texture_float4_3D, posX, posY, posZ);

    // Store the result in the output array at the calculated index
    output[idx] = result;
}