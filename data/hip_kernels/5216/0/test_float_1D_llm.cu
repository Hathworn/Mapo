#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

texture<float,  1, hipReadModeElementType> texture_float_1D;
texture<float,  2, hipReadModeElementType> texture_float_2D;
texture<float,  3, hipReadModeElementType> texture_float_3D;

texture<float4, 1, hipReadModeElementType> texture_float4_1D;
texture<float4, 2, hipReadModeElementType> texture_float4_2D;
texture<float4, 3, hipReadModeElementType> texture_float4_3D;

extern "C"

// Optimized kernel utilizing threadIdx to allow for potential parallelism
__global__ void test_float_1D(float *output, float posX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (idx == 0) // Ensure only one thread writes the output
    {
        float result = tex1D(texture_float_1D, posX);
        output[0] = result;
    }
}