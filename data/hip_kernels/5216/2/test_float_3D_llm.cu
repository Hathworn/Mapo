#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

texture<float,  1, hipReadModeElementType> texture_float_1D;
texture<float,  2, hipReadModeElementType> texture_float_2D;
texture<float,  3, hipReadModeElementType> texture_float_3D;

texture<float4, 1, hipReadModeElementType> texture_float4_1D;
texture<float4, 2, hipReadModeElementType> texture_float4_2D;
texture<float4, 3, hipReadModeElementType> texture_float4_3D;

extern "C"

__global__ void test_float_3D(float *output, float posX, float posY, float posZ)
{
    // Optimize by leveraging memory coalescing for output
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        float result = tex3D(texture_float_3D, posX, posY, posZ);
        output[idx] = result;
    }
}