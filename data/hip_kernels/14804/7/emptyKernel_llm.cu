#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel with potential workload allocation
__global__ void optimizedKernel(int *data, int num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit for threads that are out of bounds
    if (idx >= num_elements) return;

    // Process data if within bounds (Example workload)
    data[idx] = idx * 2;
}