#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void printThreadIDs() {
    // Use a single printf to reduce overhead
    printf("\n threadIdx.x : %d, threadIdx.y : %d", threadIdx.x, threadIdx.y);
}