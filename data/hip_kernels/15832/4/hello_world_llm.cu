#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_world() {
    // Use thread synchronization for better performance
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Hello CUDA world \n");
    }
}