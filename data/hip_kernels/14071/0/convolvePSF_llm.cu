#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

__global__ void convolvePSF(int width, int height, int imageCount, short *image, short *results, float *psf, int psfRad, int psfDim)
{
    // Calculate global thread coordinates
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread coordinates are within image bounds
    if (x >= width || y >= height) return;

    // Calculate convolution boundaries
    const int minX = max(x - psfRad, 0);
    const int minY = max(y - psfRad, 0);
    const int maxX = min(x + psfRad + 1, width);
    const int maxY = min(y + psfRad + 1, height);
    const int dx = maxX - minX;
    const int dy = maxY - minY;

    // Shared memory for convolution area
    __shared__ float convArea[13][13];

    // Initialize sum and correction factors
    float sum = 0.0;
    int xCorrection = x - psfRad < 0 ? 0 : psfDim - dx;
    int yCorrection = y - psfRad < 0 ? 0 : psfDim - dy;

    // Load image segment into shared memory and calculate sum
    for (int i = 0; i < dx; ++i) {
        for (int j = 0; j < dy; ++j) {
            float value = float(image[0 * width * height + (minX + i) * height + minY + j]);
            sum += value;
            convArea[i][j] = value;
        }
    }

    // Calculate the sum of differences
    float sumDifference = 0.0;
    for (int i = 0; i < dx; ++i) {
        for (int j = 0; j < dy; ++j) {
            sumDifference += fabsf(convArea[i][j] / sum - psf[(i + xCorrection) * psfDim + j + yCorrection]);
        }
    }

    // Write result to the global memory
    results[0 * width * height + x * height + y] = int(1000.0 * sumDifference);
}