#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_cmp_kernel(std::size_t n, int* aptr, int* bptr, int* rptr) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n && aptr[i] < bptr[i]) {
        atomicMax(rptr, 1);  // Use atomic operation for thread safety
    }
}