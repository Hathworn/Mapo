#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    float *feats = features + frame * ldf;

    // Apply lifter coefficients if necessary
    if (cepstral_lifter != 0.0f) {
        for (int c = thread_id; c < num_cols; c += blockDim.x) {
            feats[c] *= lifter_coeffs[c];
        }
    }

    // Use __syncthreads() to ensure lifter application is completed before energy floor is applied
    __syncthreads();

    // Only first thread in each block applies energy floor
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = logf(energy_floor);

        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        feats[0] = energy;
    }
}