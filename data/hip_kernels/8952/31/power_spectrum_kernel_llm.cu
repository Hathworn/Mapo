#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x; // Unique global thread ID
    float *Ar = A_in + blockIdx.x * ldi;
    float *Aw = A_out + blockIdx.x * ldo;
    
    int half_length = row_length / 2;
    int grid_size = blockDim.x * gridDim.x; // Total number of threads

    for (int idx = thread_id; idx < half_length; idx += grid_size) {
        // Ignore special case
        if (idx == 0) continue;

        float2 val = reinterpret_cast<float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret);
    }

    // Process special case by thread 0 of each block
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}