#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NUMBER_OF_BLOCKS 256
#define NUMBER_OF_THREADS 64

__device__ double dotProduct(double *a, double *b, int size) {
    double result = 0;
    for(int i = 0; i < size; i++) {
        result += a[i] * b[i];
    }
    return result;
}

__device__ int getElementsPerUnit(int total, int number_of_units) {
    int elements_per_unit = total / number_of_units;
    double remains = total % number_of_units;
    if(remains != 0) {
        elements_per_unit += 1;
    }
    return elements_per_unit;
}

__global__ void dotProductKernel(double *a, double *b, int a_size, int b_size, double *results, int vector_size) {
    int a_vectors_per_block = getElementsPerUnit(a_size, gridDim.x);
    int b_vectors_per_thread = getElementsPerUnit(b_size, blockDim.x);

    int a_start = blockIdx.x * a_vectors_per_block;
    int a_end = min(a_start + a_vectors_per_block, a_size); // Optimize with min

    int b_start = threadIdx.x * b_vectors_per_thread;
    int b_end = min(b_start + b_vectors_per_thread, b_size); // Optimize with min

    for(int a_index = a_start; a_index < a_end; a_index++) {
        for(int b_index = b_start; b_index < b_end; b_index++) {
            results[a_index * b_size + b_index] = dotProduct(&a[a_index * vector_size], &b[b_index * vector_size], vector_size);
        }
    }
}