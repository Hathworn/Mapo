#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NUMBER_OF_BLOCKS 256
#define NUMBER_OF_THREADS 64

// ==========
// Macro taken from:
// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
__device__ int getElementsPerUnit(int total, int number_of_units) {
    int elements_per_unit = total / number_of_units;
    double remains = total % number_of_units;

    if(remains != 0) {
        elements_per_unit += 1;
    }

    return elements_per_unit;
}

__global__ void cosineSimilarityKernel(double *dot_products, int a_size, int b_size, double *a_norms, double *b_norms, double *results) {
    int a_vectors_per_block = getElementsPerUnit(a_size, gridDim.x);
    int b_vectors_per_thread = getElementsPerUnit(b_size, blockDim.x);

    int a_start = blockIdx.x * a_vectors_per_block;
    int a_end = min(a_start + a_vectors_per_block, a_size);  // Use min to simplify boundary check

    int b_start = threadIdx.x * b_vectors_per_thread;
    int b_end = min(b_start + b_vectors_per_thread, b_size); // Use min to simplify boundary check

    for(int a_index = a_start; a_index < a_end; a_index++) {
        for(int b_index = b_start; b_index < b_end; b_index++) {
            double a_norm = a_norms[a_index];  // Cache a_norm
            double b_norm = b_norms[b_index];  // Cache b_norm
            results[a_index * b_size + b_index] = dot_products[a_index * b_size + b_index] / (a_norm * b_norm);
        }
    }
}