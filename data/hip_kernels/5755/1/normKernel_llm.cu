#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NUMBER_OF_BLOCKS 256
#define NUMBER_OF_THREADS 64

__device__ int getElementsPerUnit(int total, int number_of_units) {
    int elements_per_unit = total / number_of_units;
    double remains = total % number_of_units;

    if(remains != 0) {
        elements_per_unit += 1;
    }

    return elements_per_unit;
}

__global__ void normKernel(double *vectors, int size, double *results, int vector_size) {
    int vectors_per_block = getElementsPerUnit(size, gridDim.x);

    // Get flat thread id
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;

    // Distribute workload among threads
    for(int vec_index = thread_id; vec_index < size; vec_index += total_threads) {
        double sum = 0.0;
        for(int i = 0; i < vector_size; i++) {
            sum += vectors[vec_index * vector_size + i] * vectors[vec_index * vector_size + i];
        }
        results[vec_index] = sqrt(sum); // Calculate norm
    }
}