#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_B( float *g_data, int dimx, int dimy )
{
    int id  = blockIdx.x * blockDim.x + threadIdx.x;

    // Load value from global memory
    float value = g_data[id];

    // Use ternary operator to eliminate branching
    value += sqrtf((id % 2 ? logf(value) : cosf(value)) + 1.f);

    // Store result back in global memory
    g_data[id] = value;
}