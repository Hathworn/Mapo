#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_C(float* _g_data, int dimx, int dimy) {
    // Use shared memory for values being accessed by multiple threads
    extern __shared__ float2 shared_data[];

    float2* g_data = reinterpret_cast<float2*>(_g_data);
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Load global memory to shared memory
    shared_data[threadIdx.x] = g_data[id];

    // Ensure all threads have loaded their data
    __syncthreads();

    // Process data from shared memory
    float2 value = shared_data[threadIdx.x];
    value.x += sqrtf(cosf(value.x) + 1.f);
    value.y += sqrtf(logf(value.y) + 1.f);

    // Write back to global memory from shared memory
    g_data[id] = value;
}