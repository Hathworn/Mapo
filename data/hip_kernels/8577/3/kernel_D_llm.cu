#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_D(float* _g_data, int dimx, int dimy) {
    float4* g_data = reinterpret_cast<float4*>(_g_data);

    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data from global to register for efficiency
    float4 value = g_data[id];

    // Optimize computation by storing repeated computations in variables
    float cos_val_x = cosf(value.x);
    float log_val_y = logf(value.y);
    float cos_val_z = cosf(value.z);
    float log_val_w = logf(value.w);

    // Perform calculations
    value.x += sqrtf(cos_val_x + 1.f);
    value.y += sqrtf(log_val_y + 1.f);
    value.z += sqrtf(cos_val_z + 1.f);
    value.w += sqrtf(log_val_w + 1.f);

    // Write results back to global memory
    g_data[id] = value;
}