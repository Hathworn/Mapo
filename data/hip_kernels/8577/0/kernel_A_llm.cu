#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_A( float *g_data, int dimx, int dimy )
{
    int ix  = blockIdx.x;
    int iy  = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = iy * dimx + ix;

    // Ensure index is within array bounds
    if (ix < dimx && iy < dimy) {
        float value = g_data[idx];

        // Conditional operations based on ix parity
        float calculation = sqrtf(((ix % 2) ? logf(value) : cosf(value)) + 1.f);
        value += calculation;

        g_data[idx] = value;
    }
}