#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    // Initialize shared buffer
    reduction_buffer[idx] = 0; 
    __syncthreads();

    // Efficient reduction using warp shuffle
    float tmp = 0;
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    atomicAdd(&reduction_buffer[0], tmp);
    __syncthreads();
    
    // Barrier to ensure reduction is complete
    tmp = reduction_buffer[0];
    
    // Compute gradient with improved memory access pattern
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}