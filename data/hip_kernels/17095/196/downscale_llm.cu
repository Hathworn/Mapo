#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimization of translate_idx_inv function
__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3; 
    ii = ii / d3;
    z = ii % d2; 
    ii = ii / d2;
    y = ii % d1; 
    ii = ii / d1;
    x = ii;
    
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

// Optimization of translate_idx function
__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3; 
    ii = ii / d3;
    z = ii % d2; 
    ii = ii / d2;
    y = ii % d1; 
    ii = ii / d1;
    x = ii;
    
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ __forceinline__ size_t idx(const size_t nc, const size_t height, const size_t width, const size_t y, const size_t x) {
    return (nc * height + y) * width + x;
}

// Kernel function optimization
__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * (blockDim.x * gridDim.x);
    if (ii >= no_elements) 
        return;

    int ipidx_base = translate_idx_inv(ii, d1, d2, d3, scale_factor, 0, 0);
    for (int i = 0; i < scale_factor; i++) {
        for (int j = 0; j < scale_factor; j++) {
            // Reduce function calls by calculating base index once
            int ipidx = ipidx_base + i * d3 * scale_factor + j;
            gradInput_data[ii] += gradOutput_data[ipidx];
        }
    }
}