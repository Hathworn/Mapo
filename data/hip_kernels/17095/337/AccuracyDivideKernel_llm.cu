#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Optimize by using blockIdx.x == 0, threadIdx.x == 0 to execute once
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        *accuracy /= N;
    }
}