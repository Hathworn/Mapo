#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize for potential performance improvements with larger workloads
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Map to unique thread index across blocks
    c[i] = a[i] + b[i];
}