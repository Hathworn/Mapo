#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"



//hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);

__global__ void kernel_example(int *c, const int *a, const int *b)
{
int i = threadIdx.x;
c[i] = a[i] + b[i];
}