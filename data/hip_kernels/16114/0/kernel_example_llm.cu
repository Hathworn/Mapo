#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel with grid-stride loop and coalesced access
__global__ void kernel_example(int *c, const int *a, const int *b, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < N; i += stride) {
        c[i] = a[i] + b[i];
    }
}