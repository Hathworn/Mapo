#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSquaredDiff(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEls) { // Only compute if idx is within bounds
        dest[idx] = (a[idx] - b[idx]) * (a[idx] - b[idx]);
    }
}