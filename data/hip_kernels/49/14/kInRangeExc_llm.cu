#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kInRangeExc(float* gData, float lower, float upper, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride-based loop for better memory coalescing and fewer launch overheads
    for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x) {
        target[i] = (gData[i] > lower) && (gData[i] < upper);
    }
}