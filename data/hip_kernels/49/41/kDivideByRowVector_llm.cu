#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivideByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total number of elements
    const unsigned int totalElements = width * height;
    
    // Loop through elements with a stride of total threads
    for (unsigned int i = idx; i < totalElements; i += gridDim.x * blockDim.x) {
        // Efficient division with __fdividef
        tgtMat[i] = __fdividef(mat[i], vec[i % width]);
    }
}