#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSubtractFromScalar(float* gData, float scalar, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    // Use stride loop optimization for efficient memory access
    for (unsigned int i = idx; i < numElements; i += stride) {
        target[i] = scalar - gData[i];
    }
}