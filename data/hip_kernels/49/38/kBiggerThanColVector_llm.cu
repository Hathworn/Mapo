#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBiggerThanColVector(float* mat, float* vec, float* tgtMat, const int width, const int height) {
    // Calculate 2D indices for matrix to improve memory access pattern
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int yStride = blockDim.y * gridDim.y;
    
    for (unsigned int y = blockIdx.y * blockDim.y + threadIdx.y; y < height; y += yStride) {
        if (x < width) { // Boundary check for width
            const unsigned int i = y * width + x;
            tgtMat[i] = mat[i] > vec[y];
        }
    }
}