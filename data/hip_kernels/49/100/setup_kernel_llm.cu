#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState* state, unsigned long long seed) {
    // Calculate unique global index for each thread
    unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize RNG state with unique sequence number
    hiprand_init(seed, tidx, 0, &state[tidx]);
}