#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBiggerThanRowVector(float* mat, float* vec, float* tgtMat, const int width, const int height) {
    // Calculate the row and column based on thread and block indices
    const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (col < width) {
        for (unsigned int row = 0; row < height; ++row) {
            unsigned int i = row * width + col;
            tgtMat[i] = mat[i] > vec[col];
        }
    }
}