#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numCols, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;

    // Ensure all threads within a block are utilized
    for (uint i = eidx; i < numElements; i += DP_BLOCKSIZE * gridDim.x) {
        shmem[threadIdx.x] += a[i] * b[i];
    }
    
    __syncthreads();

    // Use loop unrolling for warp reduction
    for (uint stride = DP_BLOCKSIZE / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}