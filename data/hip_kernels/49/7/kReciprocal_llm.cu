#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kReciprocal(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Improve memory coalescing by ensuring continuous access pattern
    for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x) {
        float data = gData[i]; // Load data into register
        target[i] = 1.0f / data; // Perform reciprocal and store to global memory
    }
}