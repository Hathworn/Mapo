#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void k_copy_reshape_rowmajor(unsigned int numEls, unsigned int a_nd, const float * a_data, const int * a_dim, const int * a_str, unsigned int z_nd, float * z_data, const int * z_dim, const int * z_str)
{
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    
    // Loop over all elements assigned to this thread
    for (unsigned int i = idx; i < numEls; i += numThreads)
    {
        const float * a_i = a_data;
        int a_ii = i;

        // Calculate the corresponding position in the source array
        for (unsigned int _d = 0; _d < a_nd; ++_d) {
            unsigned int d = a_nd - 1 - _d;
            int a_i_d = a_ii % a_dim[d];
            a_ii /= a_dim[d];
            a_i += a_i_d * a_str[d];
        }

        float * z_i = z_data;
        int z_ii = i;

        // Calculate the corresponding position in the destination array
        for (unsigned int _d = 0; _d < z_nd; ++_d) {
            unsigned int d = z_nd - 1 - _d;
            int z_i_d = z_ii % z_dim[d];
            z_ii /= z_dim[d];
            z_i += z_i_d * z_str[d];
        }

        z_i[0] = a_i[0]; // Copy data from source to destination
    }
}