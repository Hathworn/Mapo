#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kBiggerThanScalar(float* gData, float scalar, float* target, unsigned int numElements) {
    // Calculate unique global thread ID
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure each thread operates on its own index and processes remaining elements in a stride
    for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x) {
        target[i] = gData[i] > scalar;
    }
}