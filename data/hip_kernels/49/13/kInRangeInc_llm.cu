#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kInRangeInc(float* gData, float lower, float upper, float* target, unsigned int numElements) {
    // Calculate unique thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Loop with controlled stride
    for (unsigned int i = idx; i < numElements; i += stride) {
        // Direct assignment with conditional expression
        target[i] = (gData[i] >= lower && gData[i] <= upper) ? 1.0f : 0.0f;
    }
}