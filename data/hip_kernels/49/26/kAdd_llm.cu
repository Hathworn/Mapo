#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd(float* a, float* b, float* dest, unsigned int numEls, float scaleA, float scaleB) {
    // Calculate the global index using the block and thread size
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop over data with stride of total number of threads
    for (unsigned int i = idx; i < numEls; i += gridDim.x * blockDim.x) {
        // Perform scaled addition
        dest[i] = scaleA * a[i] + scaleB * b[i];
    }
}
```
