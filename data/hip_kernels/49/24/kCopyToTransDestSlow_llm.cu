#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline int getTransArrayIndex(unsigned int width, unsigned int height, unsigned int i) {
    return height * (i % width) + i / width;
}

__global__ void kCopyToTransDest(float* srcStart, float* destStart, unsigned int srcCopyWidth, unsigned int srcJumpWidth, unsigned int destJumpHeight, unsigned int numElements) {
    // Use cooperative memory access and avoid branching
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numElements) return;

    unsigned int row = idx / srcCopyWidth;
    unsigned int col = idx % srcCopyWidth;
    unsigned int destIdx = getTransArrayIndex(srcCopyWidth, destJumpHeight, idx);
    unsigned int srcIdx = row * srcJumpWidth + col;

    destStart[destIdx] = srcStart[srcIdx];
}