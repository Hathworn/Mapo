#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCopyToTransDestFast(float* srcStart, float* destStart, unsigned int srcCopyWidth, unsigned int srcCopyHeight, unsigned int srcJumpSize, unsigned int destJumpSize) {
    const unsigned int idxX = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idxY = blockIdx.y * blockDim.y + threadIdx.y;

    if (idxX < srcCopyWidth && idxY < srcCopyHeight) { // Ensure only valid threads participate
        const unsigned int srcReadIdx = idxY * srcJumpSize + idxX;
        const unsigned int destWriteIdx = idxX * destJumpSize + idxY;

        __shared__ float smem[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE + 1];

        smem[threadIdx.y][threadIdx.x] = srcStart[srcReadIdx]; // Optimize access by reversing indices
        __syncthreads();

        destStart[destWriteIdx] = smem[threadIdx.x][threadIdx.y]; // Write back using reversed indices
    }
}