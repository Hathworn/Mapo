#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBiggerThan(float* gMat1, float* gMat2, float* gMatTarget, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Add grid-stride loop to process more elements if necessary
    for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x) {
        gMatTarget[i] = gMat1[i] > gMat2[i];
    }
}