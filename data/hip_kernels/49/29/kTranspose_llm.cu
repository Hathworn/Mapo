#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTranspose(float* a, float* dest, int width, int height) {
    // Calculate global indices
    const int bx = blockIdx.x * blockDim.x;
    const int by = blockIdx.y * blockDim.y;
    const int tx = bx + threadIdx.x;
    const int ty = by + threadIdx.y;

    // Declare shared memory with padding to avoid bank conflicts
    __shared__ float smem[ADD_BLOCK_SIZE][ADD_BLOCK_SIZE + 1];
    
    // Load data into shared memory only if within bounds
    if (tx < width && ty < height) {
        smem[threadIdx.y][threadIdx.x] = a[ty * width + tx];
    }
    __syncthreads();

    // Transpose block and write to the destination matrix if within bounds
    if (by + threadIdx.x < height && bx + threadIdx.y < width) {
        dest[(bx + threadIdx.y) * height + (by + threadIdx.x)] = smem[threadIdx.x][threadIdx.y];
    }
}