#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogistic1(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    // Unroll the loop to improve performance
    for (unsigned int i = idx; i < numElements; i += stride) {
        target[i] = (1 + tanhf(gData[i] / 2)) / 2;
    }
}