#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogistic2(float* gData, float* target, unsigned int numElements) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (idx < numElements) {
        // Perform computation directly
        target[idx] = 1 / (1 + expf(-gData[idx]));
    }
}