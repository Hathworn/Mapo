#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSign(float* gData, float* target, unsigned int numElements) {
    // Calculate global index once outside the loop
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use stride in loop condition
    for (unsigned int i = idx; i < numElements; i += stride) {
        target[i] = (gData[i] > 0) - (gData[i] < 0);
    }
}