#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kEqualsRowVector(float* mat, float* vec, float* tgtMat, const int width, const int height) {
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate global index
    if (x < width && y < height) {
        unsigned int idx = y * width + x;
        tgtMat[idx] = mat[idx] == vec[x];
    }
}