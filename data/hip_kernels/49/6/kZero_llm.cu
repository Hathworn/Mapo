#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kZero(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop for better performance and reduce loop overhead
    for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x * 2) {
        target[i] = 0;
        if (i + blockDim.x * gridDim.x < numElements) {
            target[i + blockDim.x * gridDim.x] = 0;
        }
    }
}