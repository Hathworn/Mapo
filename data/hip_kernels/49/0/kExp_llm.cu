#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kExp(float* gData, float* target, unsigned int numElements) {
    // Use shared memory to reduce global memory accesses
    extern __shared__ float sharedData[];
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numElements; i += stride) {
        sharedData[threadIdx.x] = gData[i];
        __syncthreads();  // Ensure sharedData is updated

        target[i] = __expf(sharedData[threadIdx.x]);
        __syncthreads();  // Ensure all threads have written before next iteration
    }
}