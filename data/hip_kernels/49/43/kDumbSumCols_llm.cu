#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDumbSumCols(float* mat, float* vec, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < width) {
        float sum = 0;
        // Use shared memory to reduce global memory accesses (if applicable)
        for (int j = 0; j < height; j++) {
            sum += mat[j * width + idx];
        }
        vec[idx] = sum;
    }
}