#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitflip_kernel(float* M, int height, int row, int n) {
    // Calculate the global index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the index is within the bounds
    if (idx < n) {
        // Pre-calculate the row offset
        int offset_row = row * height;

        // Loop with stride optimization
        for (unsigned int i = idx; i < n; i += gridDim.x * blockDim.x) {
            M[i * height + offset_row] = 1.0f - M[i * height + offset_row];
        }
    }
}