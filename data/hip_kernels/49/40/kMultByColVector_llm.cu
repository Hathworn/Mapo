#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    const unsigned int col = idx % width; // Calculate column index
    const unsigned int row = idx / width; // Calculate row index
    
    // Process using grid-stride loop
    for (unsigned int i = row; i < height; i += blockDim.x * gridDim.x) {
        unsigned int elementIdx = i * width + col;
        tgtMat[elementIdx] = mat[elementIdx] * vec[i];
    }
}