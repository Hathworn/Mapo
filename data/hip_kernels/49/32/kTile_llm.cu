#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const int srcWidth, const int srcHeight, const int tgtWidth, const int tgtHeight) {
    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total number of threads
    const int numThreads = blockDim.x * gridDim.x;
    // Iterate over elements with stride of numThreads
    for (unsigned int i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Calculate target indices
        const int y = i / tgtWidth;
        const int x = i % tgtWidth;
        // Calculate source indices using modulo for tiling
        const int srcY = y % srcHeight;
        const int srcX = x % srcWidth;
        // Directly map tiling from src to tgt
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}