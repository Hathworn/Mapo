#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float computeDeterminant(float e00, float e01, float e02, float e10, float e11, float e12, float e20, float e21, float e22) {
    return e00 * e11 * e22 - e00 * e12 * e21 + e10 * e21 * e02 - e10 * e01 * e22 + e20 * e01 * e12 - e20 * e11 * e02;
}

__global__ void hessianKernel(float *d_output, const float *d_gxx, const float *d_gxy, const float *d_gxz, const float *d_gyy, const float *d_gyz, const float *d_gzz, float sigma, int imageW, int imageH, int imageD) {
    int n_blocks_per_width = imageW / blockDim.x;
    int z = blockIdx.x / n_blocks_per_width; // Utilize integer division
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = (blockIdx.x % n_blocks_per_width) * blockDim.x + threadIdx.x; // Use modulus for x calculation
    int i = z * imageW * imageH + y * imageW + x;

    if (x >= imageW || y >= imageH || z >= imageD) return; // Boundary check

    // Preload negative gradients for efficiency
    float a0 = -d_gxx[i], b0 = -d_gxy[i], c0 = -d_gxz[i], e0 = -d_gyy[i], f0 = -d_gyz[i], k0 = -d_gzz[i];

    // Oliver K. Smith eigenvalue algorithm
    float m = (a0 + e0 + k0) / 3.f;
    float q = computeDeterminant(a0 - m, b0, c0, b0, e0 - m, f0, c0, f0, k0 - m) / 2.f;
    float p = ((a0 - m) * (a0 - m) + 2 * (b0 * b0 + c0 * c0 + f0 * f0) + (e0 - m) * (e0 - m) + (k0 - m) * (k0 - m)) / 6.f;
    float sqrt_p = sqrt(p); // Compute square root of p once
    float phi = atan(sqrt_p * sqrt_p * sqrt_p - q * q) / (3.f * q);
    if (phi < 0) phi += 3.14159f / 3.f;

    // Calculate eigenvalues
    float cos_phi = cos(phi);
    float sin_phi = sin(phi);
    float eig1 = m + 2.f * sqrt_p * cos_phi;
    float eig2 = m - sqrt_p * (cos_phi + sqrt(3.0f) * sin_phi);
    float eig3 = m - sqrt_p * (cos_phi - sqrt(3.0f) * sin_phi);

    // Determine the maximum eigenvalue and update the output
    float max_eig = fmaxf(fmaxf(eig1, eig2), eig3);
    d_output[i] = max_eig * sigma * sigma;
}