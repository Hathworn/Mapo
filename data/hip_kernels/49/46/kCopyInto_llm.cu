#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCopyInto(float* images, float* targets, const int imgSize, const int paddingSize, const int numImages) {
    const int imgIdx = blockIdx.y * gridDim.x + blockIdx.x;
    if (imgIdx < numImages) {
        const int targetSize = imgSize + 2 * paddingSize;
        images += imgIdx * imgSize * imgSize;
        targets += imgIdx * targetSize * targetSize + paddingSize * targetSize + paddingSize;

        // Loop unrolling for better performance
        for (int y = threadIdx.y; y < imgSize; y += 16) {
            int imgOffset = y * imgSize;
            int targetOffset = y * targetSize;
            for (int x = threadIdx.x; x < imgSize; x += 16) {
                targets[targetOffset + x] = images[imgOffset + x];
            }
        }
    }
}