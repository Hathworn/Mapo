#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivide(float* a, float* b, float* dest, unsigned int numEls) {
    // Use shared memory to load elements for coalesced access
    __shared__ float sharedA[256];
    __shared__ float sharedB[256];
    
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Load data into shared memory
    if (idx < numEls) {
        sharedA[threadIdx.x] = a[idx];
        sharedB[threadIdx.x] = b[idx];
    }
    __syncthreads();

    // Perform division using shared memory
    if (idx < numEls) {
        for (unsigned int i = idx; i < numEls; i += numThreads) {
            dest[i] = __fdividef(sharedA[threadIdx.x], sharedB[threadIdx.x]);
        }
    }
}