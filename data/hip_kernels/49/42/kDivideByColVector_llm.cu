#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivideByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y; // Calculate row index
    const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x; // Calculate column index

    if (row < height && col < width) { // Ensure within matrix bounds
        const unsigned int index = row * width + col; // Compute linear index
        tgtMat[index] = __fdividef(mat[index], vec[row]); // Perform division
    }
}