#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float computeDeterminant(float e00, float e01, float e02, float e10, float e11, float e12, float e20, float e21, float e22) {
    return e00 * e11 * e22 - e00 * e12 * e21 + e10 * e21 * e02 - e10 * e01 * e22 + e20 * e01 * e12 - e20 * e11 * e02;
}

__global__ void hessianKernelO(float *d_output, float *d_output_theta, float *d_output_phi, const float *d_gxx, const float *d_gxy, const float *d_gxz, const float *d_gyy, const float *d_gyz, const float *d_gzz, float sigma, int imageW, int imageH, int imageD) {
    int n_blocks_per_width = imageW / blockDim.x;
    int z = blockIdx.x / n_blocks_per_width;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = (blockIdx.x - z * n_blocks_per_width) * blockDim.x + threadIdx.x;
    int i = z * imageW * imageH + y * imageW + x;

    if (x >= imageW || y >= imageH || z >= imageD)
        return;  // Boundary condition check

    float a0, b0, c0, d0, e0, f0;
    a0 = -d_gxx[i]; b0 = -d_gxy[i]; c0 = -d_gxz[i];
    d0 = -d_gyy[i]; e0 = -d_gyz[i]; f0 = -d_gzz[i];

    float m = (a0 + d0 + f0) / 3;
    float p = (a0 - m) * (a0 - m) + 2 * b0 * b0 + 2 * c0 * c0 + (d0 - m) * (d0 - m) + 2 * e0 * e0 + (f0 - m) * (f0 - m);
    p = p / 6;
    
    float q = computeDeterminant(a0 - m, b0, c0, b0, d0 - m, e0, c0, e0, f0 - m) / 2;
    float phi = (1.f / 3.f) * atan2(sqrtf(p * p * p - q * q), q);
    
    float cosPhi = cos(phi), sinPhi = sin(phi);
    float sqrtP = sqrtf(p);

    float eig1 = m + 2 * sqrtP * cosPhi;
    float eig2 = m - sqrtP * (cosPhi + sqrtf(3.0f) * sinPhi);
    float eig3 = m - sqrtP * (cosPhi - sqrtf(3.0f) * sinPhi);

    // Use fmaxf for selecting the maximum eigenvalue
    float maxEig = fmaxf(fmaxf(eig1, eig2), eig3);
    d_output[i] = maxEig * sigma * sigma;

    // Compute the eigenvector
    float l = maxEig / (sigma * sigma);
    a0 -= l;
    d0 -= l;
    f0 -= l;

    float xv = b0 * e0 - c0 * d0;
    float yv = e0 * a0 - c0 * b0;
    float zv = d0 * a0 - b0 * b0;
    float radius = sqrtf(xv * xv + yv * yv + zv * zv);
    float thetav = atan2f(yv, xv);

    float phiv = 0;
    if(radius > 1e-6f)
        phiv = acosf(zv / radius);

    d_output_theta[i] = thetav;
    d_output_phi[i] = phiv;
}