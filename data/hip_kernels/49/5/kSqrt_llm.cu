#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSqrt(float* gData, float* target, unsigned int numElements) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory for better memory access pattern
    extern __shared__ float sharedData[];
    
    if (idx < numElements) {
        sharedData[threadIdx.x] = gData[idx];
        __syncthreads();

        // Perform the computation
        target[idx] = sqrtf(sharedData[threadIdx.x]);
    }
}