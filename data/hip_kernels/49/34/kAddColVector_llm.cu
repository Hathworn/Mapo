#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddColVector(float* mat, float* vec, float* tgtMat, const unsigned int width, const unsigned int height, const float scaleVec) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Process multiple elements per thread to reduce global memory accesses
    for (unsigned int i = idx; i < width * height; i += stride) {
        tgtMat[i] = mat[i] + scaleVec * vec[i / width];
    }
}