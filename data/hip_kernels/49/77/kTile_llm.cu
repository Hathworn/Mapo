#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Get total number of threads
    const int numThreads = blockDim.x * gridDim.x;
    
    // Loop over all target elements for this thread
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Calculate 2D indices for tgt
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        // Map 2D indices to src
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Assign the value from src to tgt
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}