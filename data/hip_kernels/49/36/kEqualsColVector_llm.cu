#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel using shared memory for vector accessing
__global__ void kEqualsColVector(float* mat, float* vec, float* tgtMat, const int width, const int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    __shared__ float sharedVec[1024]; // assuming max threads per block is 1024

    if (threadIdx.x < height) { // each thread loads one element of vec into shared memory
        sharedVec[threadIdx.x] = vec[threadIdx.x];
    }
    __syncthreads();

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        const unsigned int row = i / width;
        tgtMat[i] = mat[i] == sharedVec[row];
    }
}