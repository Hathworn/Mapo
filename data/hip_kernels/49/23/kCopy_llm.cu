#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCopy(float* srcStart, float* destStart, const int copyWidth, const int srcJumpWidth, const int destJumpWidth, const int numElements) {
    // Calculate thread's linear index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x; 

    // Use stride loop to access multiple elements
    for (int i = idx; i < numElements; i += blockDim.x * gridDim.x) {
        int row = i / copyWidth;
        int col = i % copyWidth;
        
        // Copy from source to destination using precomputed row and col
        destStart[row * destJumpWidth + col] = srcStart[row * srcJumpWidth + col];
    }
}