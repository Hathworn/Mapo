#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for comparing elements with a scalar.
__global__ void kSmallerThanScalar(float* gData, float scalar, float* target, unsigned int numElements) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop for better occupancy
    for (unsigned int i = idx; i < numElements; i += blockDim.x * gridDim.x) {
        target[i] = gData[i] < scalar;
    }
}