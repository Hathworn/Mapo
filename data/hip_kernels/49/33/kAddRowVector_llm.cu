#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height, float scaleVec) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    const unsigned int numElements = width * height;

    // Process using stride loop to cover entire data set
    for (unsigned int i = idx; i < numElements; i += numThreads) {
        tgtMat[i] = mat[i] + scaleVec * vec[i % width];
    }
}