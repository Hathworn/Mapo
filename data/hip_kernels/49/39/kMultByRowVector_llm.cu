#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int colIdx = idx % width;
    const unsigned int rowIdx = idx / width;

    // Optimize by eliminating redundant division and modulus calculations
    if (rowIdx < height) {
        tgtMat[idx] = mat[idx] * vec[colIdx];
    }
}