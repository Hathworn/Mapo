#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLog(float* gData, float* target, unsigned int numElements) {
    // Calculate the global index for the thread
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop through elements by stride
    for (unsigned int i = idx; i < numElements; i += gridDim.x * blockDim.x) {
        target[i] = __logf(gData[i]); // Compute the log for each element
    }
}