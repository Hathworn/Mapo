#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMult(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Loop unrolling for performance improvement
    for (unsigned int i = idx; i < numEls; i += stride * 4) {
        if (i < numEls) dest[i] = a[i] * b[i];
        if (i + stride < numEls) dest[i + stride] = a[i + stride] * b[i + stride];
        if (i + 2 * stride < numEls) dest[i + 2 * stride] = a[i + 2 * stride] * b[i + 2 * stride];
        if (i + 3 * stride < numEls) dest[i + 3 * stride] = a[i + 3 * stride] * b[i + 3 * stride];
    }
}