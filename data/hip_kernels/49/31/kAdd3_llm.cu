#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd3(float* a, const float* b, const float* c, const unsigned int numEls, const float scaleA, const float scaleB, const float scaleC) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Use shared memory for temporary storage
    extern __shared__ float tempA[];

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        // Pre-fetch in a shared memory buffer
        tempA[threadIdx.x] = a[i];

        // Perform computation using shared memory 
        a[i] = scaleA * tempA[threadIdx.x] + scaleB * b[i] + scaleC * c[i];
    }
}