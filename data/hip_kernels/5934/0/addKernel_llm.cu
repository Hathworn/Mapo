#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Use stride for loop unrolling
    for (; i < N; i += stride) { // Process multiple elements per thread
        c[i] = a[i] + b[i];
    }
}