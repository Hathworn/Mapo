#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Ensure only one thread performs the division for better parallel performance
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}