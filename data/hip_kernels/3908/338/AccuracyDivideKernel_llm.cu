#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use atomic operations to ensure thread safety for updating `accuracy`
    atomicExch(accuracy, *accuracy / N);
}