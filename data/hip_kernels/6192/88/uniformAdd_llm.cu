#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) {
    __shared__ float uni;
    // Load the uniform value only in the first thread
    if (threadIdx.x == 0) {
        uni = uniforms[blockIdx.x + blockOffset];
    }

    // Improved address calculation using atomic operations
    unsigned int address = (blockIdx.x * (blockDim.x << 1)) + baseIndex + threadIdx.x;

    __syncthreads(); // Ensure shared data is ready before proceeding

    // Avoid redundant computation with conditional logic
    g_data[address] += uni;
    if ((threadIdx.x + blockDim.x) < n) {
        g_data[address + blockDim.x] += uni;
    }
}