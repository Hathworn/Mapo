#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load accuracy into register for faster access
    float acc = *accuracy;

    // Divide in register
    acc /= N;

    // Write back to global memory
    *accuracy = acc;
}