#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii /= d3;
    z = ii % d2;
    ii /= d2;
    y = ii % d1;
    ii /= d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii /= d3;
    z = ii % d2;
    ii /= d2;
    y = ii % d1;
    ii /= d1;
    x = ii;
    w /= scale_factor;
    z /= scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ __forceinline__ size_t idx(const size_t nc, const size_t height, const size_t width, const size_t y, const size_t x) {
    return (nc * height + y) * width + x;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    // Calculate unique global index for each thread
    long ii = threadIdx.x + blockDim.x * blockIdx.x + 
              (threadIdx.y + blockDim.y * blockIdx.y) * (blockDim.x * gridDim.x);
    
    // Exit if index exceeds number of elements
    if (ii >= no_elements) return;
    
    int baseIpIdx = translate_idx_inv(ii, d1, d2, d3, scale_factor, 0, 0);
    
    // Unroll loops to exploit compiler optimizations
    float sum = 0.0f;
    for (int i = 0; i < scale_factor; i++) {
        for (int j = 0; j < scale_factor; j++) {
            int ipidx = baseIpIdx + j * d3 + i * d3 * d2;
            sum += gradOutput_data[ipidx];
        }
    }
    
    // Atomic add to prevent race condition
    atomicAdd(&gradInput_data[ii], sum);
}