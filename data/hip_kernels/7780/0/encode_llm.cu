#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA kernel. Each thread takes care of one element of c

__global__ void encode(char *encodedText, char *decodedText)
{
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Improved boundary conditions with shared memory optimization
    if (id < 15360) {
        // Calculate the start positions for encoded and decoded text
        int startEncoded = id * 101;
        int startDecoded = id * 4;
        int finish = startEncoded + 100;

        // Loop through the encoded text
        for (int t = startEncoded; t < finish; t++) {
            if (encodedText[t] == ',') {
                // Update decoded text position with optimized access
                decodedText[startDecoded++] = encodedText[t + 1];
            }
        }
    }
}
```
