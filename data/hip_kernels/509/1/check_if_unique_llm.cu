#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void check_if_unique(const unsigned *keys, unsigned *is_unique, size_t kSize) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified the ID calculation
    if (id < kSize) { // Combined condition to check all within bounds
        is_unique[id] = (id == 0 || keys[id] != keys[id - 1]) ? 1 : 0; // Optimized indexing and unified condition
    }
}