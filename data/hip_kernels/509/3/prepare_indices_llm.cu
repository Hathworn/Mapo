#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prepare_indices(const unsigned num_keys, unsigned *data) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (index < num_keys) {
        data[index] = index;
    }
}