#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_from_gpu()
{
    // Use a single thread to reduce redundant printf calls
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Hello World from the GPU!\n");
    }
}