#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel by reducing printf usage in large grid dimensions
__global__ void hello_from_gpu()
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID
    printf("Hello World from block %d and thread %d!\n", blockIdx.x, threadIdx.x); // Reduced to a single printf
}