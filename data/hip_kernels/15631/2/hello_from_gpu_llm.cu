#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_from_gpu()
{
    // Use built-in variable for combined thread and block index
    int idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    // Improved printf to reduce overhead per thread
    printf("Hello World from block-%d and linear thread-%d!\n", blockIdx.x, idx);
}