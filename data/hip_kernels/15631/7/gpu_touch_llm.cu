#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_touch(uint64_t *x, const size_t size)
{
    // Calculate the stride for the entire grid
    const size_t stride = blockDim.x * gridDim.x;

    // Use a stride loop to ensure all elements are touched
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += stride)
    {
        x[i] = 0; // Zero-initialize the array element
    }
}