#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_from_gpu()
{
    // Use block and thread indices for distinguishing output in a real use-case
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello World from the GPU! Thread ID: %d\n", idx);
}