#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate the global index for the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Check if the index is within bounds
    if(i >= N) return;
    
    // Decompose the index to get the input spatial, layer, and batch indices
    int in_s = i % spatial;
    int rem = i / spatial;
    int in_c = rem % layers;
    int b = rem / layers;

    // Calculate the input and output indices based on the forward flag
    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;

    // Perform the forward or backward operation
    if (forward) 
        out[i2] = x[i1];
    else 
        out[i1] = x[i2];
}