#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out) {
    // Calculate unique global index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * (gridDim.x * blockDim.x);
    if (i >= N) return; // Early exit for out-of-bound threads
    
    int in_index = i;
    int in_w = in_index % w;
    int in_h = (in_index / w) % h;
    int in_c = (in_index / (w * h)) % c;
    int b = in_index / (w * h * c);
    
    int out_c = c / (stride * stride);
    
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    if (forward) {
        out[out_index] = x[in_index];
    } else {
        out[in_index] = x[out_index];
    }
}