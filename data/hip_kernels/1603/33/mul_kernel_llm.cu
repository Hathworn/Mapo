#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global thread index efficiently
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use N and idx instead of i for clarity
    if (idx < N) {
        // Perform multiplication only for valid indices
        Y[idx * INCY] *= X[idx * INCX];
    }
}