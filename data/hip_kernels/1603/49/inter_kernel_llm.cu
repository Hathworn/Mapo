#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int i = idx + idy * gridDim.x * blockDim.x;

    // Calculate which block 'b' and within-block position 'j'
    if(i < (NX + NY) * B){
        int b = i / (NX + NY);
        int j = i % (NX + NY);

        // Efficiently accessing with branch-free ternary operator
        OUT[i] = (j < NX) ? X[b * NX + j] : Y[b * NY + j - NX];
    }
}