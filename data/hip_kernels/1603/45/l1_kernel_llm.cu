#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l1_kernel(int n, const float *pred, const float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;  

    if (i < n) {
        // Store the difference
        float diff = truth[i] - pred[i];
        
        // Compute error and delta in a single step
        error[i] = fabsf(diff);
        delta[i] = (diff > 0) ? 1.0f : -1.0f;
    }
}