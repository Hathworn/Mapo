#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int w_size = w * size;
    int wc = w * c;

    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (wc * h);

    int out_index = j + w * (i + h * (k + b * c));

    int half_size = size / 2;
    int w_offset = -half_size;
    int h_offset = -half_size;

    int cur_h, cur_w, index;
    bool valid;

    #pragma unroll
    for(int l = 0; l < size; ++l){
        #pragma unroll
        for(int m = 0; m < size; ++m){
            cur_h = i + h_offset + l;
            cur_w = j + w_offset + m;
            index = cur_w + w * (cur_h + h * (k + b * c));

            // Check boundaries
            valid = (cur_h >= 0 && cur_h < h && cur_w >= 0 && cur_w < w);
            
            // Reduce conditional offset to a single operation
            delta[out_index] += valid ? rate * (x[index] - x[out_index]) : 0;
        }
    }
}