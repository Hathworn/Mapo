#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    // Initialize sum for this thread
    float thread_sum = 0.0f;
    
    // Perform accumulation
    for(int k = 0; k < n; ++k)
    {
        thread_sum += x[k * groups + i];
    }
    
    // Write the result to global memory
    sum[i] = thread_sum;
}