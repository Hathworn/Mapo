#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int spatial_size = w * h; // Calculate spatial size outside loop
    int k = id % c;
    int b = id / c;

    float sum = 0.0f; // Use a local variable for sum
    int out_index = k + c * b;
    int offset = h * w * (k + b * c); // Precompute input offset
    for(int i = 0; i < spatial_size; ++i){
        sum += input[offset + i]; // Use offset in accessing input
    }
    output[out_index] = sum / spatial_size; // Store the result directly
}