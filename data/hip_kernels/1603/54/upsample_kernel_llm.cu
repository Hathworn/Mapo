#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;

    // Unrolling the division reduces arithmetic operations
    int out_w = index % (w * stride);
    int out_h = (index / (w * stride)) % (h * stride);
    int out_c = (index / (w * stride * h * stride)) % c;
    int b = index / (w * stride * h * stride * c);

    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_index = b * w * h * c + out_c * w * h + in_h * w + in_w;

    // Use single expression for conditional operation
    if (forward) {
        out[index] += scale * x[in_index];
    } else {
        atomicAdd(x + in_index, scale * out[index]);
    }
}