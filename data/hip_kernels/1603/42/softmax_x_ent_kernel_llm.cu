#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global index of the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Ensure we don't access out of bounds
    if(i < n){
        float t = truth[i];
        float p = pred[i];

        // Use ternary operator to compute values efficiently
        error[i] = (t != 0.0f) ? -logf(p) : 0.0f;
        delta[i] = t - p;
    }
}