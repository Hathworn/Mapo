#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Calculate the unique global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure valid index
    if (i < n) {
        // Use ternary operation for binarization
        binary[i] = (x[i] >= 0) ? 1.0f : -1.0f;
    }
}