#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    // Calculate the global index properly using block and thread indices.
    int index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Return early if the thread index is out of the output array boundaries
    if (index >= n * size * batch) return;

    // Use the fast integer division and modulo operations
    int i = index % size;
    index /= size;
    int j = index % n;
    index /= n;
    int k = index;

    // Perform the bias addition
    output[(k * n + j) * size + i] += biases[j];
}