#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Precompute global index and stride
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop to handle larger data sets
    for (int i = index; i < n; i += stride) {
        float diff = truth[i] - pred[i];
        error[i] = diff * diff; // Compute squared error
        delta[i] = diff;        // Compute delta
    }
}