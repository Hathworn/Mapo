#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop over the data to maximize occupancy and cover larger data sizes
    for (int i = idx; i < N; i += stride) {
        float val = X[i * INCX];
        if((val * val) < (ALPHA * ALPHA)) {
            X[i * INCX] = 0;
        }
    }
}