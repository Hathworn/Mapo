#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    local[id] = 0;
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) { // Distribute work evenly among threads
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += delta[index]; // Each thread handles its portion
        }
    }

    __syncthreads();

    if (id == 0) {
        float sum = 0.0f;
        for (int i = 0; i < threads; ++i) {
            sum += local[i]; // Accumulate results
        }
        mean_delta[filter] = sum * (-1.f / sqrtf(variance[filter] + 0.00001f)); // Compute mean_delta
    }
}