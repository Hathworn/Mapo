#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    // Use block indexing and thread indexing for optimal memory access
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Iterate over elements using stride for coalesced memory access
    for (int i = idx; i < n; i += blockDim.x * gridDim.x)
    {
        if (mask[i] == mask_num) // Condition to check mask
        {
            x[i] *= scale; // Scale if condition is met
        }
    }
}