#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    float h = hsv.x;
    float s = hsv.y;
    float v = hsv.z;

    float r, g, b;
    if (s == 0) {
        r = g = b = v; // Achromatic case
    } else {
        int index = static_cast<int>(h);
        float f = h - index;
        float p = v * (1 - s);
        float q = v * (1 - s * f);
        float t = v * (1 - s * (1 - f));

        // Use a lookup table for branch elimination
        float3 cases[6] = {
            make_float3(v, t, p),
            make_float3(q, v, p),
            make_float3(p, v, t),
            make_float3(p, q, v),
            make_float3(t, p, v),
            make_float3(v, p, q)
        };
        float3 result = cases[index % 6]; // Handle wrap-around
        r = result.x;
        g = result.y;
        b = result.z;
    }

    // Clamp results between 0 and 1
    r = fminf(fmaxf(r, 0.0f), 1.0f);
    g = fminf(fmaxf(g, 0.0f), 1.0f);
    b = fminf(fmaxf(b, 0.0f), 1.0f);
    return make_float3(r, g, b);
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    float r = rgb.x;
    float g = rgb.y;
    float b = rgb.z;

    float h, s, v;
    float max = fmaxf(r, fmaxf(g, b));
    float min = fminf(r, fminf(g, b));
    float delta = max - min;
    v = max;

    if (max == 0) {
        s = 0; 
        h = -1; // Undefined hue
    } else {
        s = delta / max;
        if (r == max) {
            h = (g - b) / delta;
        } else if (g == max) {
            h = 2 + (b - r) / delta;
        } else {
            h = 4 + (r - g) / delta;
        }
        if (h < 0) h += 6;
    }
    return make_float3(h, s, v);
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int size = batch * w * h;
    int id = (blockDim.x * blockIdx.x) + threadIdx.x;
    if (id >= size) return;

    int pixelIndex = id % w;
    id /= w;
    int rowIndex = id % h;
    id /= h;

    float rshift = rand[0];
    float gshift = rand[1];
    float bshift = rand[2];
    float r0 = rand[8 * id + 0];
    float r1 = rand[8 * id + 1];
    float r2 = rand[8 * id + 2];
    float r3 = rand[8 * id + 3];

    saturation = r0 * (saturation - 1) + 1;
    saturation = (r1 > 0.5f) ? (1.f / saturation) : saturation;
    exposure = r2 * (exposure - 1) + 1;
    exposure = (r3 > 0.5f) ? (1.f / exposure) : exposure;

    size_t offset = id * h * w * 3;
    image += offset;
    float r = image[pixelIndex + w * (rowIndex + h * 0)];
    float g = image[pixelIndex + w * (rowIndex + h * 1)];
    float b = image[pixelIndex + w * (rowIndex + h * 2)];
    float3 rgb = make_float3(r, g, b);

    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }

    image[pixelIndex + w * (rowIndex + h * 0)] = rgb.x * scale + translate + (rshift - 0.5f) * shift;
    image[pixelIndex + w * (rowIndex + h * 1)] = rgb.y * scale + translate + (gshift - 0.5f) * shift;
    image[pixelIndex + w * (rowIndex + h * 2)] = rgb.z * scale + translate + (bshift - 0.5f) * shift;
}