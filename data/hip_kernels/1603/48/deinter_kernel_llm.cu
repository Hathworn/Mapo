#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Compute global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if within valid bounds
    int totalElements = (NX + NY) * B;
    if(i >= totalElements) return;

    // Calculate block and element indices
    int b = i / (NX + NY);
    int j = i % (NX + NY);

    // Update corresponding array based on index
    if (j < NX) {
        if (X) {
            // Atomic addition to prevent race conditions
            atomicAdd(&X[b * NX + j], OUT[i]);
        }
    } else {
        if (Y) {
            // Atomic addition to prevent race conditions
            atomicAdd(&Y[b * NY + j - NX], OUT[i]);
        }
    }
}