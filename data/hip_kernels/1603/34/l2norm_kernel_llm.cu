#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;

    int b = index / spatial;
    int i = index % spatial;

    // Allocate shared memory to store sum for spatial block
    extern __shared__ float shared_sum[];
    int thread_id = threadIdx.x;
    shared_sum[thread_id] = 0;

    // Calculate the sum of squares using shared memory for improved efficiency
    for (int f = 0; f < filters; ++f) {
        int index = b * filters * spatial + f * spatial + i;
        shared_sum[thread_id] += powf(x[index], 2);
    }

    __syncthreads();

    // Reduce the sum within the block
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (thread_id < stride) {
            shared_sum[thread_id] += shared_sum[thread_id + stride];
        }
        __syncthreads();
    }

    float sum = sqrtf(shared_sum[0]);
    if (sum == 0) sum = 1;

    // Normalize using the computed sum
    for (int f = 0; f < filters; ++f) {
        int index = b * filters * spatial + f * spatial + i;
        x[index] /= sum;
        dx[index] = (1 - x[index]) / sum;
    }
}