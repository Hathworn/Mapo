#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll the loop for efficiency
    int gridSize = blockDim.x * gridDim.x;
    while (i < N) {
        X[i * INCX] *= ALPHA;
        i += gridSize;
    }
}