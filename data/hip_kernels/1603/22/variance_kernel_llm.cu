#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1.f / (batch * spatial - 1);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float var = 0.0f;

    // Loop through batch and spatial dimensions to calculate variance
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            // Accumulate variance using shared computation of differences
            float diff = x[index] - mean[i];  // Compute once per element
            var += diff * diff;  // More efficient multiplication
        }
    }

    variance[i] = var * scale;  // Efficient final scaling
}