#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;
    // Unroll loop to improve performance
    for(int j = 0; j < batch; ++j){
        for(int i = 0; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i + id;

            // Use conditional increment to avoid branch divergence
            if(i+id < spatial) {
                local[id] += delta[index] * (x[index] - mean[filter]);
            }
        }
    }

    __syncthreads();

    // Reduce within the block using parallel reduction
    if (id < 32) {
        for (int offset = threads / 2; offset > 0; offset /= 2) {
            local[id] += __shfl_down_sync(0xffffffff, local[id], offset);
        }
    }

    if (id == 0) {
        variance_delta[filter] = local[0] * (-.5f) * powf(variance[filter] + .00001f, (float)(-3.f/2.f));
    }
}