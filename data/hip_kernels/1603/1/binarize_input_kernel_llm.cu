#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;

    // Use shared memory to reduce redundant global memory accesses
    extern __shared__ float shared_mean[];

    float mean = 0.0f;
    int i;

    // Calculate partial mean in each thread
    for (i = 0; i < n; ++i) {
        mean += fabsf(input[i * size + s]);
    }

    // Store mean in shared memory for further access
    shared_mean[threadIdx.x] = mean;

    // Synchronize all threads to ensure shared memory is fully populated
    __syncthreads();

    // Calculate global mean by accumulating shared memory values
    if (threadIdx.x == 0) {
        float total_mean = 0.0f;
        for (i = 0; i < blockDim.x; ++i) {
            total_mean += shared_mean[i];
        }
        mean = total_mean / (n * blockDim.x);
    }

    // Broadcast the calculated mean to all threads in the block
    __syncthreads();

    // Use the mean from shared memory
    if (threadIdx.x == 0) {
        shared_mean[0] = mean;
    }

    // Synchronize to ensure all threads have the correct mean value
    __syncthreads();

    // Binarize input using the shared mean
    for (i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? shared_mean[0] : -shared_mean[0];
    }
}