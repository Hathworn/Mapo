#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    // Calculate global thread index
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Use shared memory to optimize mean calculation
    extern __shared__ float shared_weights[];
    float mean = 0;

    // Load weights into shared memory
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[i] = weights[f*size + i];
    }
    __syncthreads();

    // Compute mean using shared memory
    for (int i = 0; i < size; ++i) {
        mean += fabsf(shared_weights[i]);
    }
    mean = mean / size;
    
    // Assign binary values based on the mean
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f*size + i] = (shared_weights[i] > 0) ? mean : -mean;
    }
}