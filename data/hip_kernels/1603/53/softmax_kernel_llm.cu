#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;

    // Find the largest element
    for (i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = fmaxf(val, largest);
    }

    // Compute the exponentials and sum them up
    for (i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp);
        sum += e;
        output[i * stride] = e;
    }

    // Normalize the output
    for (i = 0; i < n; ++i) {
        output[i * stride] /= sum;
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Use blockIdx.x directly
    if (id >= batch * groups) return;

    int b = id / groups;
    int g = id % groups;
    softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}