#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (; index < batch * n * n; index += stride) {
        int f1 = index / n;
        int f2 = index % n;
        if (f2 <= f1) continue;

        float sum = 0;
        float norm1 = 0;
        float norm2 = 0;

        // Unroll loop for efficiency
        for (int i = 0; i < size; ++i) {
            int i1 = f1 * size + i;
            int i2 = f2 * size + i;
            sum += output[i1] * output[i2];
            norm1 += output[i1] * output[i1];
            norm2 += output[i2] * output[i2];
        }

        norm1 = sqrt(norm1);
        norm2 = sqrt(norm2);
        float norm = norm1 * norm2;
        sum = sum / norm;

        // Unroll loop for efficiency
        for (int i = 0; i < size; ++i) {
            int i1 = f1 * size + i;
            int i2 = f2 * size + i;
            delta[i1] += -scale * sum * output[i2] / norm;
            delta[i2] += -scale * sum * output[i1] / norm;
        }
    }
}