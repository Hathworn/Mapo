#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float sum = 0;
    int num_elements = batch * spatial;
    int offset = i * spatial;

    // Unroll the loop to enhance performance
    for (int index = 0; index < num_elements; index += spatial) {
        #pragma unroll
        for (int k = 0; k < spatial; ++k) {
            sum += delta[index + offset + k];
        }
    }

    mean_delta[i] = sum * (-1.f / sqrtf(variance[i] + .00001f));
}