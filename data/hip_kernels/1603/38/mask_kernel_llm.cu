#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop for improved efficiency; works for larger datasets
    int stride = blockDim.x * gridDim.x;

    // Loop with stride to cover all elements
    for (; i < n; i += stride) {
        if (mask[i] == mask_num) {
            x[i] = val;
        }
    }
}