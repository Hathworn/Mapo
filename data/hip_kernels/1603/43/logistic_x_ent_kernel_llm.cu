#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate a global thread index using block and thread indices
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a strided loop to cover all elements in case grid size is smaller than n
    for (; i < n; i += blockDim.x * gridDim.x) {
        float t = truth[i];
        float p = pred[i];
        
        // Keep calculations outside of conditional checks for optimization
        float p_eps = p + 0.0000001f;
        float inv_p_eps = 1 - p + 0.0000001f;
        
        // Compute error using updated terms
        error[i] = -t * log(p_eps) - (1 - t) * log(inv_p_eps);
        
        // Calculate delta value
        delta[i] = t - p;
    }
}