#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Access only every 'INCX' element
    int index = i * INCX;

    // Write ALPHA to the specified position if within bounds
    if(index < N) X[index] = ALPHA;
}