#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Prepare for reduction variable
    float sum = 0.0f;
    
    // Use registers for repetitive calculations and loop unrolling for optimization
    int spatial_stride = i * spatial;
    float variance_offset = variance[i] + .00001f;
    float inverse_variance = (__powf(variance_offset, -1.5f)) * -.5f;

    for (int j = 0; j < batch; ++j)
    {
        int batch_stride = j * filters * spatial;
        for (int k = 0; k < spatial; ++k)
        {
            int index = batch_stride + spatial_stride + k;
            sum += delta[index] * (x[index] - mean[i]);
        }
    }
    
    // Apply computed variance_multiplier
    variance_delta[i] = sum * inverse_variance;
}