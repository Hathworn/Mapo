#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int i, b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    for(b = 0; b < batch; ++b) {
        for(i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index] * x_norm[index];  // Use p as offset directly in the loop
        }
    }
    part[p] = sum;
    __syncthreads();

    if (p == 0) {
        float block_sum = 0;  // Local reduction within the block
        for(i = 0; i < BLOCK; ++i) block_sum += part[i];
        atomicAdd(&scale_updates[filter], block_sum);  // Atomic addition to ensure correct accumulation in global memory
    }
}