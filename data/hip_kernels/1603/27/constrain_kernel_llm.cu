#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop for optimal memory access
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        X[idx * INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[idx * INCX]));
    }
}