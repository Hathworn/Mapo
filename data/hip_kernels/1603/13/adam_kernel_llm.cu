#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;  // Optimize index calculation
    if (index >= N) return;

    float B1_t_pow = powf(B1, t);  // Reduce redundant computations by reusing the power result
    float B2_t_pow = powf(B2, t);  // Reduce redundant computations by reusing the power result

    float mhat = m[index] / (1.f - B1_t_pow);
    float vhat = v[index] / (1.f - B2_t_pow);

    x[index] = x[index] + rate * mhat / (sqrtf(vhat) + eps);
}