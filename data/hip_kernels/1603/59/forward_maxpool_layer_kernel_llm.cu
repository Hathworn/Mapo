#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    // Calculate grid-specific threads.
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;
    
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int c = in_c;

    // Calculate input image positions.
    int j = id % w;
    id /= w; 
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    // Initialize for searching the maximum value.
    int out_index = j + w * (i + h * (k + c * b));
    float max = -INFINITY;
    int max_i = -1;

    // Use registers for loop invariants and boundaries.
    int start_h = h_offset + i * stride;
    int start_w = w_offset + j * stride;
    
    // Traverse filter bounds.
    for(int l = 0; l < size; ++l) {
        int cur_h = start_h + l;
        if (cur_h < 0 || cur_h >= in_h) continue; // Skip out-of-bound h
        
        for(int m = 0; m < size; ++m) {
            int cur_w = start_w + m;
            if (cur_w < 0 || cur_w >= in_w) continue; // Skip out-of-bound w
            
            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            float val = input[index];

            // Check for new maximum.
            if (val > max) {
                max = val;
                max_i = index;
            }
        }
    }
    
    // Store results in the output.
    output[out_index] = max;
    indexes[out_index] = max_i;
}