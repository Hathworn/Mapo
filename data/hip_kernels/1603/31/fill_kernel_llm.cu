#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Optimize thread index calculation
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Loop to cover array with a stride
    for (; i < N; i += stride)
    {
        X[i * INCX] = ALPHA;
    }
}