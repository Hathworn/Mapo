#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Using grid-stride loop to enhance parallelism
    for(int b = 0; b < batch; ++b){
        for(int i = p; i < size; i += BLOCK){
            int index = i + size * (filter + n * b);
            sum += delta[index];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Reduce sum within block using a single shared memory loop
    if (p < BLOCK / 2) {
        for (int stride = BLOCK / 2; stride > 0; stride >>= 1) {
            if (p < stride) part[p] += part[p + stride];
            __syncthreads();
        }
    }

    // Only the first thread updates the bias
    if (p == 0) bias_updates[filter] += part[0];
}