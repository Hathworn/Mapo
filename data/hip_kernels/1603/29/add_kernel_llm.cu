#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index more directly
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int stride = blockDim.x * gridDim.x;

    // Use strided loop for better performance on large arrays
    for (; i < N; i += stride) {
        X[i * INCX] += ALPHA;
    }
}