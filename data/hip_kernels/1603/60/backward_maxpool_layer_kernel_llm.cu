#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes) {
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int c = in_c;
    int area = (size - 1) / stride;

    int id = blockIdx.x * blockDim.x + threadIdx.x; // Flattened block and thread index for 1D grid
    if (id >= n) return;

    int j = id % in_w;
    int i = (id / in_w) % in_h;
    int k = (id / (in_w * in_h)) % in_c;
    int b = id / (in_c * in_w * in_h);

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    float d = 0.0f;

    // Unrolling the loop to improve performance
    #pragma unroll
    for (int l = -area; l <= area; ++l) {
        int out_h = (i - h_offset) / stride + l;
        if (out_h < 0 || out_h >= h) continue; // Check valid range for out_h only once

        #pragma unroll
        for (int m = -area; m <= area; ++m) {
            int out_w = (j - w_offset) / stride + m;
            if (out_w < 0 || out_w >= w) continue; // Check valid range for out_w only once

            int out_index = out_w + w * (out_h + h * (k + c * b));
            if (indexes[out_index] == id) { // Reduce branching and computation
                d += delta[out_index];
            }
        }
    }

    prev_delta[id] += d; // Update previous delta
}