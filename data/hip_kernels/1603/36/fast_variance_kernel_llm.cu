#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];
    int id = threadIdx.x;
    local[id] = 0;
    int filter = blockIdx.x;
    
    // Unroll the loop for performance improvement
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += powf((x[index] - mean[filter]), 2);
        }
    }
    
    __syncthreads();
    
    // Use parallel reduction to accumulate results
    for (unsigned int stride = threads / 2; stride > 0; stride >>= 1) {
        __syncthreads();
        if (id < stride) {
            local[id] += local[id + stride];
        }
    }
    
    if (id == 0) {
        variance[filter] = local[0] / (spatial * batch - 1);
    }
}