#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplified 1D index calculation
    int stride = blockDim.x * gridDim.x; // Use stride for loop
    for (int i = idx; i < N; i += stride) { // Loop to cover all elements
        Y[i * INCY] = powf(X[i * INCX], ALPHA); // Use powf for single-precision
    }
}