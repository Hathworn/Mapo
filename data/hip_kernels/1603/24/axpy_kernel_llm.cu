#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Use one-dimensional grid for simplicity
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Loop to load balance among available threads
    for(int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        Y[OFFY + idx * INCY] += ALPHA * X[OFFX + idx * INCX];
    }
}