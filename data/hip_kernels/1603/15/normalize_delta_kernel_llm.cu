#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Updated kernel with improved indexing and efficiency
__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < N) { // Ensure within bounds
        int f = (index / spatial) % filters;

        // Cache frequent calculations
        float inv_sqrt_var = 1.f / sqrtf(variance[f] + .00001f);
        float norm_factor = 2.f / (spatial * batch);

        // Update delta with optimized computation
        delta[index] = delta[index] * inv_sqrt_var 
                       + variance_delta[f] * norm_factor * (x[index] - mean[f]) 
                       + mean_delta[f] * norm_factor;
    }
}