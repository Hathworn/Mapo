#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified thread index calculation for 1D grid
    int stride = gridDim.x * blockDim.x;  // Calculate stride for loop unrolling 

    for(int i = id; i < size; i += stride) {  // Use loop unrolling for better performance
        input[i] = (rand[i] < prob) ? 0 : input[i] * scale;
    }
}