#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use early exit to improve efficiency
    if (i >= n) return;

    // Streamline computation by ensuring only one memory fetch per element
    float si = s[i];
    c[i] = si * a[i] + (1.0f - si) * (b ? b[i] : 0.0f);
}