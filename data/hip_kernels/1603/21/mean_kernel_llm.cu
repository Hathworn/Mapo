#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Calculate the scale factor
    float scale = 1.f / (batch * spatial);

    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid out-of-bound access
    if (i >= filters) return;

    // Initialize mean value
    float sum = 0;

    // Efficiently accumulate values
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += x[index];
        }
    }

    // Store the scaled mean
    mean[i] = sum * scale;
}