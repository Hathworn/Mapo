#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global index using blockIdx.x
    int stride = gridDim.x * blockDim.x; // Calculate the total number of threads processing across all blocks

    // Loop over the data with a stride to process multiple elements per thread
    for (; i < N; i += stride) {
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
    }
}