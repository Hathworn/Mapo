#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Use 1D grid and block for better indexing
    if(id >= n) return;

    int out_area = w * h;  // Pre-calculate area
    int k = id % c;
    int b = id / c;  // Avoid unnecessary div-mod operations in the loop below

    float scale = out_delta[k + c * b] / out_area;  // Pre-calculate division
    int base_index = h * w * (k + b * c);  // Pre-calculate base index

    for(int i = 0; i < out_area; ++i){
        in_delta[base_index + i] += scale;  // Directly use calculated scale value
    }
}