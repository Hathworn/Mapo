#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized weighted_delta_kernel using shared memory
__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure no out-of-bound access
    if (i >= n) return;

    // Use local variables for repeated array access
    float dc_i = dc[i];
    float s_i = s[i];

    if (da) da[i] += dc_i * s_i;
    if (db) db[i] += dc_i * (1 - s_i);

    float a_i = a[i];
    float b_i = b[i];
    ds[i] += dc_i * (a_i - b_i);
}