#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (index < N) { // Simplified boundary check
        int f = (index / spatial) % filters;
        float mean_f = mean[f];
        float variance_f = variance[f];
        float x_value = x[index];

        // Pre-compute and reuse mean_f and variance_f
        x[index] = (x_value - mean_f) / (sqrtf(variance_f + 1e-5f)); // Adjusted epsilon notation for clarity
    }
}