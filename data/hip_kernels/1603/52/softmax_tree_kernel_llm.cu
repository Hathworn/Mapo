#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;

    // Use a single loop to find the largest value and calculate exponentials
    for (i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = fmaxf(val, largest);
        float e = expf(val / temp - largest / temp);
        sum += e;
        output[i * stride] = e;
    }

    // Compute final output by normalizing with the sum
    float invSum = 1.0f / sum; // Compute reciprocal of sum
    for (i = 0; i < n; ++i) {
        output[i * stride] *= invSum; // Optimize division by multiplication
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Optimize id calculation
    if (id >= spatial * batch * groups) return;

    int s = id % spatial;
    int g = (id / spatial) % groups; // Optimize id calculation
    int b = id / (spatial * groups); // Calculate batch

    int goff = group_offset[g] * spatial;
    int boff = b * stride;
    
    // Launch softmax_device using the calculated offsets
    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}