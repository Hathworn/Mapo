#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate global thread index for 1D grid
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    // Process all elements in this block
    while (i < n) { 
        c[i] += a[i] * b[i]; // Perform operation
        i += blockDim.x * gridDim.x; // Move to the next segment
    }
}