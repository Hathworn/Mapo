#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_forwardElimination( float * fullMatrix, float * B, unsigned int nComp ) {
    unsigned int t = threadIdx.x;
    unsigned int baseIndex = t * nComp * nComp;

    for (unsigned int i = 0; i < nComp - 1; i++) {
        float pivot = fullMatrix[baseIndex + i * nComp + i]; // Cache pivot value
        for (unsigned int j = i + 1; j < nComp; j++) {
            float div = fullMatrix[baseIndex + j * nComp + i] / pivot; // Use cached pivot value
            for (unsigned int k = 0; k < nComp; k++) {
                fullMatrix[baseIndex + j * nComp + k] -= div * fullMatrix[baseIndex + i * nComp + k];
            }
            B[baseIndex + j] -= div * B[baseIndex + i];
        }
    }
}