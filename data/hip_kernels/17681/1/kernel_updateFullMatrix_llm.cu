#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_updateFullMatrix( float * device_fullMatrix, float * B, float * V, float * Cm, float * Em, float * Rm, float dt, unsigned int nComp ) {
    // Calculate global thread index
    unsigned int myIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop to update when index is within bounds
    while (myIndex < nComp) {
        B[myIndex] = V[myIndex] * Cm[myIndex] / (dt / 2.0f) + Em[myIndex] / Rm[myIndex];
        myIndex += blockDim.x * gridDim.x; // Move to next element for this thread
    }
}