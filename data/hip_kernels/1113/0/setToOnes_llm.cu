#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setToOnes(float *data, int size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x; // 1D grid of 1D blocks
    int stride = blockDim.x * gridDim.x; // Calculate stride

    for (int i = index; i < size; i += stride) { // Iterate over data with stride
        data[i] = 1;
    }
}