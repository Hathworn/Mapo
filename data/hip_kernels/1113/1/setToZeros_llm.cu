#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setToZeros(float *data, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index (optimizing readability)
    
    // Using loop unrolling for better performance
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < size; i += stride)
    {
        data[i] = 0;
    }
}