#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calc_entropy_atomic(float *float_image_in, float *entropy_out, int blk_size) {

    // Use shared memory for intermediate sum
    __shared__ float sum;
    __shared__ float entropy;

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        sum = 0.0f;
        entropy = 0.0f;
    }
    __syncthreads();
    
    int blocksize = blk_size * blk_size;
    // Calculate pixel index offset for each block and thread
    int v_offset_to_blkrow = gridDim.x * blockDim.x * blockDim.y * blockIdx.y;
    int v_offset_to_pixrow = blockDim.x * gridDim.x * threadIdx.y;
    int h_offset = blockDim.x * blockIdx.x + threadIdx.x;
    int idx = v_offset_to_blkrow + v_offset_to_pixrow + h_offset;
    int out_idx = blockIdx.y * gridDim.x + blockIdx.x;
    
    // Normalize image
    float normalizedValue = float_image_in[idx] * float_image_in[idx] / blocksize;
    atomicAdd(&sum, normalizedValue);
    __syncthreads();
    
    normalizedValue /= sum;
    // Calculate Shannon entropy using atomic addition
    if (normalizedValue > 0) { // Ensure log is not computed for zero
        atomicAdd(&entropy, -normalizedValue * log2(normalizedValue));
    }
    
    __syncthreads();
    
    // Write entropy result to the output array
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        entropy_out[out_idx] = entropy;
    }
}