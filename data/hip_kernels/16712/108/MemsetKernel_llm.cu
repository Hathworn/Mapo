#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global indices
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Guard against out-of-bounds
    if (i < h && j < w) {
        const int pos = i * w + j;
        image[pos] = value;
    }
}