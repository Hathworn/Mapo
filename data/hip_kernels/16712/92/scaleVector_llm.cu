#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    const int stride = blockDim.x * gridDim.x;
    for (int pos = blockIdx.x * blockDim.x + threadIdx.x; pos < len; pos += stride) {
        // Compute result using multiple threads
        d_res[pos] = d_src[pos] * scale;
    }
}