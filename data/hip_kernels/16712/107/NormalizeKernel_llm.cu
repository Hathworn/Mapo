#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Optimize index calculation for i
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation for j

    if (i < h && j < w) { // Combine boundary check and processing
        const int pos = i * s + j; // Avoid recalculating 'pos'
        float scale = normalization_factor[pos];
        float invScale = 1.0f / (scale + (scale == 0.0f)); // Prevent division by zero, optimize condition
        image[pos] *= invScale; // Simplify scaling operation
    }
}