#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Load loc from global memory once
        short2 loc = loc_[ptidx];
        
        // Compute scaled values and store directly
        float scaledX = loc.x * scale;
        float scaledY = loc.y * scale;
        x[ptidx] = scaledX;
        y[ptidx] = scaledY;
    }
}