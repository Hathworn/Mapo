#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Precompute scaled coordinates and fractional parts
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    // Avoid multiple casting by computing integer coords once
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);

    float value = src[image_row_offset + j];
    
    // Use shared memory for atomic operations if using CUDA dynamic parallelism or similar
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        float weight = dx * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    tx -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        float weight = (1.0f - dx) * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    ty -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        float weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    tx += 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        float weight = dx * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}