#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // Compute partial sum for inner product in parallel
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Use warp-level reduction for the final reduction step
    if (idx < 32) { // Assuming blockDim.x is a multiple of 32 (warp size)
        for (int i = idx + 32; i < blockDim.x; i += 32) {
            tmp += reduction_buffer[i];
        }
        reduction_buffer[idx] = tmp;
    }
    __syncthreads();

    // Compute the gradient using the reduced inner product value
    if (idx == 0) {
        tmp = 0.0f;
        for (int i = 0; i < min(blockDim.x, 32); ++i) {
            tmp += reduction_buffer[i]; // Only consider first warp
        }
        reduction_buffer[0] = tmp;
    }
    __syncthreads();

    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}