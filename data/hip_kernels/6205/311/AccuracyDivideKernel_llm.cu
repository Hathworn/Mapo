#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory to minimize global memory access within the kernel
    __shared__ float shared_accuracy;
    
    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy;
    }
    __syncthreads();

    // Each thread updates a copy of the accuracy
    float local_accuracy = shared_accuracy / N;
    
    if (threadIdx.x == 0) {
        // Only one thread writes back to global memory
        *accuracy = local_accuracy;
    }
}