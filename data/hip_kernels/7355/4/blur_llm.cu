#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blur(float *input, float *output, int height, int width)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < height && y < width)
    {
        for (int k = 0; k < 3; k++)
        {
            float sum = 0.0f;
            int count = 0;
            // Use shared memory and loop unrolling for optimization
            __shared__ float input_shared[BLOCK_SIZE_X][BLOCK_SIZE_Y + 2 * BLUR_SIZE];
            
            // Load data into shared memory
            for (int i = -BLUR_SIZE; i <= BLUR_SIZE; i += BLOCK_SIZE_Y)
            {
                for (int j = -BLUR_SIZE; j <= BLUR_SIZE; j += BLOCK_SIZE_X)
                {
                    int shared_x = threadIdx.x + j;
                    int shared_y = threadIdx.y + i;

                    if (shared_x >= 0 && shared_x < width && shared_y >= 0 && shared_y < height)
                    {
                        input_shared[threadIdx.x + BLUR_SIZE][threadIdx.y + BLUR_SIZE] = input[3 * ((shared_y + x) * width + (shared_x + y)) + k];
                    }
                }
            }
            __syncthreads();
            
            // Compute blur using shared memory
            for (int i = -BLUR_SIZE; i <= BLUR_SIZE; ++i)
            {
                for (int j = -BLUR_SIZE; j <= BLUR_SIZE; ++j)
                {
                    int shared_x = threadIdx.x + BLUR_SIZE + j;
                    int shared_y = threadIdx.y + BLUR_SIZE + i;
                    
                    if (shared_x >= 0 && shared_x < width && shared_y >= 0 && shared_y < height)
                    {
                        count++;
                        sum += input_shared[shared_x][shared_y];
                    }
                }
            }
            
            output[3 * (x * width + y) + k] = sum / count;
        }
    }
}