#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define M 10000
#define N 10000

__global__ void add(int *a, int *b, int *c)
{
    // Calculate the global index. Each thread computes multiple elements.
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int j = blockDim.y * blockIdx.y + threadIdx.y;

    // Use a single index for linear processing instead of 2D indexing for better memory coalescing.
    unsigned int index = j * M + i;

    // Ensure index is within bounds before processing.
    if (i < M && j < N)
        c[index] = a[index] + b[index];
}