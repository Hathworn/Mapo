#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void rgb2gray (float * input, float *output, int height, int width)
{
    // Calculate global index for current thread
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check if thread is within image bounds
    if(x < height && y < width)
    {
        // Reduce redundant calculations by precomputing index
        unsigned int idx3 = (x * width + y) * 3;
        unsigned int idx = x * width + y;
        
        // Load color values
        float r = input[idx3];
        float g = input[idx3 + 1];
        float b = input[idx3 + 2];
        
        // Convert to grayscale using weighted sum
        output[idx] = (0.21f * r + 0.71f * g + 0.07f * b);
    }
}