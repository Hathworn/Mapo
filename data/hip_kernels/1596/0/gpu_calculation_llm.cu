#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_calculation(float c0r, float c0i, float float_step, float imag_step, int *results, unsigned n, int W, int H, int inicial) {
    // Calculate global index using block and thread identifiers
    const long unsigned globalIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if globalIndex is within bounds
    if (globalIndex < n) {
        // Determine coordinates for the complex plane
        int x = (globalIndex + inicial) / W;
        int y = (globalIndex + inicial) % H;

        // Calculate the real and imaginary parts of the complex number
        float point_r = c0r + x * float_step;
        float point_i = c0i + y * imag_step;

        const int M = 1000; // Maximum iterations
        int j = -1; // -1 indicates no failure in mandelbrot computation

        // Initial values for Mandelbrot computation
        float old_r = 0.0f; // Current real part
        float old_i = 0.0f; // Current imaginary part

        // Perform Mandelbrot iteration
        for (int i = 1; i <= M; i++) {
            // Update real and imaginary parts
            float aux = (old_r * old_r) - (old_i * old_i) + point_r;
            old_i = (2.0f * old_r * old_i) + point_i;
            old_r = aux;

            // Check if the modulus exceeds 2 (optimizing condition check)
            if ((old_r * old_r + old_i * old_i) > 4.0f) {
                j = i;
                break;
            }
        }

        // Store the result for the current complex number
        results[globalIndex] = j;
    }
}