#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized vadd kernel

__global__ void vadd(const float *a, const float *b, float *c, int n)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) // Ensure within bounds
    {
        c[i] = a[i] + b[i];
    }
}