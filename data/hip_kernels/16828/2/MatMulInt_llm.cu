#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatMulInt(int *a, int b, int *c, int ROW, int COLUMNS) {
    // Cache row and column limits to reduce global memory access
    int rowLimit = ROW, columnLimit = COLUMNS;

    // Compute linear index
    int idx = blockIdx.y * blockDim.y * columnLimit + blockIdx.x * blockDim.x + threadIdx.y * columnLimit + threadIdx.x;

    // Check bounds before computation
    if (blockIdx.x * blockDim.x + threadIdx.x < rowLimit && blockIdx.y * blockDim.y + threadIdx.y < columnLimit) {
        c[idx] = a[idx] * b; 
    }
}