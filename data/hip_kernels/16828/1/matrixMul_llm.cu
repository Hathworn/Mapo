#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMul(int *a, int *b, int *c, int ROW, int COLUMNS, int temp)
{
    // Calculate row and column index of the matrix element
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < ROW && col < COLUMNS) 
    {
        int sum = 0;
        // Accumulate the sum for dot product
        for(int i = 0; i < temp; i++)
        {
            sum += a[row * temp + i] * b[i * COLUMNS + col];
        }
        // Store the result in matrix c
        c[row * COLUMNS + col] = sum;
    }
}