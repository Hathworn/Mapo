#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized: Add two matrices using HIP
__global__ void MatAdd(int *a, int *b, int *c, int ROW, int COLUMNS) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < COLUMNS && iy < ROW) { // Corrected boundary check
        int idx = iy * COLUMNS + ix; // Moved calculation inside bounds check
        c[idx] = a[idx] + b[idx];
    }
}