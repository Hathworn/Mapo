#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float softplus_kernel(float x, float threshold = 20) {
    if (x > threshold) return x;                // Simplified condition for large values
    else if (x < -threshold) return expf(x);    // Simplified condition for small values
    return logf(expf(x) + 1);
}

__device__ float tanh_activate_kernel(float x) {
    return (2 / (1 + expf(-2 * x)) - 1);        // Calculate tanh activation
}

__global__ void mish_kernel(const float *input, float *output, int num_elem) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= num_elem) return;                // Ensure idx is within bounds

    // Utilize both softplus and tanh kernel directly for mish computation
    output[idx] = input[idx] * tanh_activate_kernel(softplus_kernel(input[idx]));
}