#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _hSigmoidKer(float const *in, float *out, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {  // Use a single if condition to improve readability and reduce branching
        float val = in[index];  // Reduce redundant memory access
        if (val > 3)
            out[index] = 1;
        else if (val < -3)
            out[index] = 0;
        else
            out[index] = (val + 3) / 6;  // Use the stored value
    }
}