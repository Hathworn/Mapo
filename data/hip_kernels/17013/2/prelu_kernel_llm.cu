#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prelu_kernel(const float *input, float *output, int num_elem, int input_size, int fm_size, const float* gamma) {

    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    // Ensure index is within bounds
    if (idx < num_elem) {

        // Pre-compute channel index
        int c = (idx % input_size) / fm_size;

        // Simplify condition and avoid branch
        float val = input[idx];
        output[idx] = (val >= 0.0f) ? val : val * gamma[c];
    }
}