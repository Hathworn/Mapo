#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void b_scan_reduce_cuda(int *values, int *cumulative, size_t bins)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;

    int nb_here = (bins - (blockDim.x * blockIdx.x) < blockDim.x) ? bins - (blockDim.x * blockIdx.x) : blockDim.x;

    __shared__ int shared_vals[1024]; // Assuming max blockDim.x
    if (tid < nb_here)
        shared_vals[tid] = values[id];
    __syncthreads();

    int next_th = 2;
    for (int threshold = 1; threshold < blockDim.x / 2; threshold <<= 1)
    {
        // Optimization: Minimize conditional checks
        if (tid < nb_here && tid - threshold >= 0 && ((nb_here - 1) - tid) % next_th == 0)
        {
            shared_vals[tid] += shared_vals[tid - threshold];
        }
        next_th <<= 1;
        __syncthreads();
    }

    if (tid == nb_here - 1)
        shared_vals[tid] = 0;
    next_th = nb_here >> 1;
    int tmp;
    for (int threshold = nb_here; threshold > 1; threshold >>= 1)
    {
        if (tid < nb_here && tid - next_th >= 0 && (nb_here - 1 - tid) % threshold == 0) {
            tmp = shared_vals[tid];
            shared_vals[tid] += shared_vals[tid - next_th];
            shared_vals[tid - next_th] = tmp;
        }
        next_th >>= 1;
        __syncthreads();
    }

    if (tid < nb_here)
        cumulative[id] = shared_vals[tid];
}