#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogram_cuda(int *histogram, float *values, size_t nb, float bin_size, float min, int bins, int nb_thread)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x) * ITEMS_PER_THREAD;
    int thread_id = threadIdx.x;

    // Use shared memory instead of malloc for better performance
    extern __shared__ int s_bins[];
    int *local_hist = s_bins + thread_id * bins;

    if (id == 0)
        printf("Bin size : %f\n", bin_size);

    // Initialize local histogram
    for (int i = 0; i < bins; i++)
        local_hist[i] = 0;

    // Compute local bins
    for (int i = 0; i < ITEMS_PER_THREAD; i++)
    {
        for (int j = 0; j < bins; j += 1)
        {
            if (id + i < nb && values[id + i] <= ((float)min + (float)(j + 1) * bin_size))
            {
                local_hist[j] += 1;
                break;
            }
        }
    }
    __syncthreads();

    // Reduce local bins into shared memory
    for (size_t s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (thread_id < s)
        {
            for (int j = 0; j < bins; j++)
            {
                local_hist[j] += s_bins[(thread_id + s) * bins + j];
            }
        }
        __syncthreads();
    }

    // Combine results to histogram
    if (thread_id == 0)
    {
        for (int i = 0; i < bins; i++) {
            histogram[i + blockIdx.x * bins] = s_bins[i];
        }
    }
}