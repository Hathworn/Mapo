#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void max_min_cuda(float *d_in1, float *d_in2, float *d_max, float *d_min, size_t nb)
{
    extern __shared__ float shared_data[];
    float* s_in1 = shared_data;
    float* s_in2 = shared_data + blockDim.x;

    int ft_id = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // Load input into shared memory
    if (ft_id < nb) {
        s_in1[tid] = d_in1[ft_id];
        s_in2[tid] = d_in2[ft_id];
    }
    else {
        s_in1[tid] = -FLT_MAX;
        s_in2[tid] = FLT_MAX;
    }
    __syncthreads();

    // Perform reduction in shared memory
    for (size_t s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && tid + s < blockDim.x) {
            s_in1[tid] = fmaxf(s_in1[tid], s_in1[tid + s]);
            s_in2[tid] = fminf(s_in2[tid], s_in2[tid + s]);
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        d_max[blockIdx.x] = s_in1[0];
        d_min[blockIdx.x] = s_in2[0];
    }
}