#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Precompute indices and strides for better performance
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;
    int o = blockIdx.x;
    int k = blockIdx.x % input_n;
    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;
    int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y*gridDim.y;

    output += o * output_w * output_h;
    input += o * input_w * input_h;
    float the_weight = weight[k];
    float the_bias = bias[k];

    // Utilize loop unrolling for faster computation
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            float sum = 0;
            float *ptr_input = input + yy * dH * input_w + xx * dW;

            #pragma unroll
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    sum += ptr_input[kx];
                }
                ptr_input += input_w; // next input line
            }

            // Calculate output using precomputed weight and bias
            output[yy * output_w + xx] = the_weight * sum + the_bias;
        }
    }
}