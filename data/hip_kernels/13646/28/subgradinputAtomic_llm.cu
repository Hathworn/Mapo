#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW) {
    // compute output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // calculate thread/block based offsets
    int o = blockIdx.x;
    int k = blockIdx.x % input_n; // updated variable usage
    int xx = threadIdx.x + blockDim.x * blockIdx.y; // unify xx_start and yy_start calculation
    int yy = threadIdx.y + blockDim.y * blockIdx.z;

    // select input/output plane with better scope management
    gradOutput += o * output_w * output_h;
    gradInput += o * input_w * input_h;

    // get weight for the current output plane
    float the_weight = weight[k];

    // limit threads within the output dimensions
    if (yy < output_h && xx < output_w) {
        float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float z = *ptr_gradOutput * the_weight;

        // loop interchange and reducing redundant pointer calculations
        for (int ky = 0; ky < kH; ky++) {
            for (int kx = 0; kx < kW; kx++) {
                atomicAdd(&(ptr_gradInput[kx]), z);
            }
            ptr_gradInput += input_w; // increment row
        }
    }
}