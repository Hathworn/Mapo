#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate unique index for each thread
    int time_index = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;

    if (time_index < output_w) {
        // Compute base pointers for gradInput, gradOutput, and indices
        float *gradInput_data = gradInput + blockIdx.x * input_w * input_n + time_index * input_n * dW;
        float *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + time_index * input_n;
        float *indices_data = indices + blockIdx.x * output_w * input_n + time_index * input_n;

        // Cache to local memory in the loop scope to improve performance
        for (int feat = 0; feat < input_n; ++feat) {
            int index = (int)indices_data[feat] * input_n + feat;
            // Use atomic operation if necessary to prevent race conditions
            atomicAdd(&gradInput_data[index], gradOutput_data[feat]);
        }
    }
}