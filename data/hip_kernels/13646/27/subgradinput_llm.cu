#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinput(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW) {
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Calculate current thread's output coordinate
    int xx = blockDim.x * blockIdx.x + threadIdx.x;
    int yy = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Early exit if thread is out of bounds
    if (xx >= output_w || yy >= output_h) return;

    // Compute offsets based on block ID
    int o = blockIdx.z; // Using blockIdx.z for batch dimension
    int k = o % input_n;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += o * input_w * input_h;

    // Get weight
    float the_weight = weight[k];

    // Compute gradInput
    float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
    float z = gradOutput[yy * output_w + xx] * the_weight;
    
    for (int ky = 0; ky < kH; ky++) {
        for (int kx = 0; kx < kW; kx++) {
            ptr_gradInput[kx] += z;  // Accumulate weighted gradOutput into gradInput
        }
        ptr_gradInput += input_w;  // Advance to the next row in gradInput
    }
}