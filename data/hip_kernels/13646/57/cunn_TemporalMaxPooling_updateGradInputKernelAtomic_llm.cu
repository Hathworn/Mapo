#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Compute global time index
    int globalTimeIdx = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    
    // Check bounds to avoid extra computation
    if (globalTimeIdx < output_w) {
        // Calculate the base data pointers for this block
        float *gradInput_data = gradInput + blockIdx.x * input_w * input_n + globalTimeIdx * input_n * dW;
        float *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + globalTimeIdx * input_n;
        float *indices_data = indices + blockIdx.x * output_w * input_n + globalTimeIdx * input_n;

        // Use loop unrolling for potential performance gain
        int feat = 0;
        for (; feat + 3 < input_n; feat += 4) {
            atomicAdd(&gradInput_data[(int)indices_data[feat] * input_n + feat], gradOutput_data[feat]);
            atomicAdd(&gradInput_data[(int)indices_data[feat+1] * input_n + feat+1], gradOutput_data[feat+1]);
            atomicAdd(&gradInput_data[(int)indices_data[feat+2] * input_n + feat+2], gradOutput_data[feat+2]);
            atomicAdd(&gradInput_data[(int)indices_data[feat+3] * input_n + feat+3], gradOutput_data[feat+3]);
        }
        // Handle the remaining elements
        for (; feat < input_n; ++feat) {
            atomicAdd(&gradInput_data[(int)indices_data[feat] * input_n + feat], gradOutput_data[feat]);
        }
    }
}