#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure threads only process elements within bounds
    if (idx < numEls) {
        dest[idx] = a[idx] + b[idx];
    }
}