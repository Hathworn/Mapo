#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;  // Use blockDim for more flexibility
    const int y = blockIdx.y * blockDim.y + threadIdx.y;  // Use blockDim for more flexibility
    if (x < width && y < height) { // Check boundary before accessing memory
        int p = y * pitch + x;     // Calculate the index after boundary check
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}