#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * blockDim.x + tx; // Simplified index calculation
    const int y = blockIdx.y * 16; // Fixed block height
    const int p_base = y * width + x;
    
    if (x < width) {
        float val = d_Data[p_base];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        minvals[tx] = FLT_MAX; // Set default values to infinity for non-participating threads
        maxvals[tx] = -FLT_MAX;
    }
    
    for (int ty = 1; ty < 16; ty++) {
        const int p = p_base + ty * width; // Linear address calculation
        if (x < width && ty + y < height) {
            float val = d_Data[p];
            minvals[tx] = fminf(minvals[tx], val);
            maxvals[tx] = fmaxf(maxvals[tx], val);
        }
    }
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tx < stride) {
            minvals[tx] = fminf(minvals[tx], minvals[tx + stride]);
            maxvals[tx] = fmaxf(maxvals[tx], maxvals[tx + stride]);
        }
        __syncthreads();
    }
    
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}