#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LowPassRowMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    __shared__ float data[CONVROW_W + 2*RADIUS];
    const int tx = threadIdx.x;
    const int block = blockIdx.x/(NUM_SCALES+3);
    const int scale = blockIdx.x - (NUM_SCALES+3)*block;
    const int xout = block*CONVROW_W + tx;
    const int loadPos = xout - RADIUS;
    const int yptr = blockIdx.y*pitch;
    const int writePos = yptr + height*pitch*scale + xout;
    const float *kernel = d_Kernel + scale*16;

    // Pre-fetched boundary condition handling
    float val;
    if (loadPos < 0)
        val = d_Data[yptr];
    else if (loadPos >= width)
        val = d_Data[yptr + width - 1];
    else
        val = d_Data[yptr + loadPos];
    data[tx] = val;
    __syncthreads();

    // Avoid out-of-bound computation with redundant threads
    if (xout < width && tx < CONVROW_W) {
        float sum = 0.0f;
        #pragma unroll
        for (int i = 0; i < 4; ++i) {
            sum += (data[tx + i] + data[tx + 8 - i]) * kernel[i];
        }
        sum += data[tx + 4] * kernel[4];
        d_Result[writePos] = sum;
    }
}