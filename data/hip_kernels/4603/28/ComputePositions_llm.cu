#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define POSBLK_SIZE 256

__global__ void ComputePositions(float *g_Data1, float *g_Data2, float *g_Data3, int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Compute global index
    if (i >= numPts)
        return;

    int p = d_Ptrs[i];
    
    float val[7];
    val[0] = g_Data2[p];
    val[1] = g_Data2[p - 1];
    val[2] = g_Data2[p + 1];
    
    float dx = 0.5f * (val[2] - val[1]);
    float dxx = 2.0f * val[0] - val[1] - val[2];
    
    val[3] = g_Data2[p - w];
    val[4] = g_Data2[p + w];
    
    float dy = 0.5f * (val[4] - val[3]);
    float dyy = 2.0f * val[0] - val[3] - val[4];
    
    val[5] = g_Data3[p];
    val[6] = g_Data1[p];
    
    float ds = 0.5f * (val[6] - val[5]);
    float dss = 2.0f * val[0] - val[5] - val[6];
    
    // Optimized to reuse values for better performance
    float gData2_pw_p1 = g_Data2[p + w + 1];
    float gData2_pw_m1 = g_Data2[p - w - 1];

    float gData3_p1   = g_Data3[p + 1];
    float gData3_m1   = g_Data3[p - 1];
    float gData1_pw   = g_Data1[p + w];
    float gData1_pw_mw = g_Data1[p - w];

    float dxy = 0.25f * (gData2_pw_p1 + gData2_pw_m1 - gData2[p - w + 1] - g_Data2[p + w - 1]);
    float dxs = 0.25f * (gData3_p1 + g_Data1[p - 1] - g_Data1[p + 1] - gData3_m1);
    float dys = 0.25f * (g_Data3[p + w] + gData1_pw_mw - g_Data3[p - w] - gData1_pw);

    float idxx = dyy * dss - dys * dys;
    float idxy = dys * dxs - dxy * dss;
    float idxs = dxy * dys - dyy * dxs;
    float idyy = dxx * dss - dxs * dxs;
    float idys = dxy * dxs - dxx * dys;
    float idss = dxx * dyy - dxy * dxy;

    float det = idxx * dxx + idxy * dxy + idxs * dxs;
    float idet = 1.0f / det;

    float pdx = idet * (idxx * dx + idxy * dy + idxs * ds);
    float pdy = idet * (idxy * dx + idyy * dy + idys * ds);
    float pds = idet * (idxs * dx + idys * dy + idss * ds);

    // Correct position for boundaries
    if (pdx < -0.5f || pdx > 0.5f || pdy < -0.5f || pdy > 0.5f || pds < -0.5f || pds > 0.5f)
    {
        pdx = __fdividef(dx, dxx);
        pdy = __fdividef(dy, dyy);
        pds = __fdividef(ds, dss);
    }

    float dval = 0.5f * (dx * pdx + dy * pdy + ds * pds);

    d_Sift[i + 0 * maxPts] = (p % w) + pdx;
    d_Sift[i + 1 * maxPts] = (p / w) + pdy;
    d_Sift[i + 2 * maxPts] = d_ConstantA[0] * exp2f(pds * d_ConstantB[0]);
    d_Sift[i + 3 * maxPts] = val[0] + dval;

    float tra = dxx + dyy;
    det = dxx * dyy - dxy * dxy;
    d_Sift[i + 4 * maxPts] = __fdividef(tra * tra, det);
}