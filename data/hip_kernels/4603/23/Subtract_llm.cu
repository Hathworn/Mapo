#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Use grid size to compute the global index
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;
    
    // Ensure only valid threads perform operations
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}