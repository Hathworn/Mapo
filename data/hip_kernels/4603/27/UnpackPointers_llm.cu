#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    // Optimize thread indexing
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    const int bDimX = blockDim.x;
    int numPts = 0;
    
    // Unroll outer loop and reduce condition checks
    for (int y = bx; y < h/32; y += gridDim.x) { // Utilize grid for outer loop
        for (int base_x = tx; base_x < w; base_x += bDimX) { // Utilize block for inner loop
            unsigned int val = minmax[y * w + base_x];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + base_x;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}