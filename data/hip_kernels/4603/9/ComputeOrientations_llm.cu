#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOrientations(float *g_Data, float *d_Sift, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];

    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Initialize histogram
    for (int i = tx; i < 13 * 32; i += 32)
        hist[i] = 0.0f;
    __syncthreads();

    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    if (tx < 15)
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));

    int xp = (int)(d_Sift[bx + 0 * maxPts] - 6.5f);
    int yp = (int)(d_Sift[bx + 1 * maxPts] - 6.5f);
    int px = xp & 15;
    int x = tx - px;

    // Load data to shared memory
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = max(0, min(w - 1, xp + x));
        int yi = max(0, min(h - 1, yp + y));
        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    // Compute gradients and build histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = (int)(16.0f * atan2f(dy, dx) / 3.1416f + 16.5f) % 32;
            float grad = sqrtf(dx * dx + dy * dy);
            atomicAdd(&hist[32 * (x - 1) + bin], grad * gauss[x] * gauss[y]);
        }
    }
    __syncthreads();

    // Combine histogram bins
    for (int y = 0; y < 5; y++)
        hist[y * 32 + tx] += hist[(y + 8) * 32 + tx];
    __syncthreads();
    for (int y = 0; y < 4; y++)
        hist[y * 32 + tx] += hist[(y + 4) * 32 + tx];
    __syncthreads();
    for (int y = 0; y < 2; y++)
        hist[y * 32 + tx] += hist[(y + 2) * 32 + tx];
    __syncthreads();
    hist[tx] += hist[32 + tx];
    __syncthreads();

    // Smooth histogram
    if (tx <= 31) {
        int next_tx = (tx + 1) & 31;
        int prev_tx = (tx + 31) & 31;
        hist[32 + tx] = 6 * hist[tx] + 4 * (hist[next_tx] + hist[prev_tx]) + (hist[(tx + 2) & 31] + hist[(tx + 30) & 31]);
    }
    __syncthreads();

    // Find peaks in histogram
    float v = hist[32 + tx];
    hist[tx] = (v > hist[32 + ((tx + 1) & 31)] && v >= hist[32 + ((tx + 31) & 31)]) ? v : 0.0f;
    __syncthreads();

    if (tx == 0) {
        float maxval1 = 0.0f, maxval2 = 0.0f;
        int i1 = -1, i2 = -1;

        // Find top two peaks
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }

        // Calculate peak location and orientation
        float val1 = hist[32 + ((i1 + 1) & 31)];
        float val2 = hist[32 + ((i1 + 31) & 31)];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Sift[bx + 5 * maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);

        if (maxval2 < 0.8f * maxval1)
            i2 = -1;
        if (i2 >= 0) {
            val1 = hist[32 + ((i2 + 1) & 31)];
            val2 = hist[32 + ((i2 + 31) & 31)];
            peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Sift[bx + 6 * maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        } else {
            d_Sift[bx + 6 * maxPts] = i2;
        }
    }
}