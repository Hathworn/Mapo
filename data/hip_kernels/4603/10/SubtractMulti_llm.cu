#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim to enhance flexibility
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Use blockDim to enhance flexibility
    if (x < width && y < height)
    {
        int sx = pitch * height; // Precompute shared variable
        int p = threadIdx.z * sx + y * pitch + x; // Use precomputed sx
        d_Result[p] = d_Data[p] - d_Data[p + sx];
    }
    // Removed unnecessary __syncthreads(), no shared memory is used
}