#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Use blockDim to calculate global x and y coordinates
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x; // Simplify index calculation
    if (x < width && y < height)
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    // Remove unnecessary __syncthreads() as there are no shared memory operations
}