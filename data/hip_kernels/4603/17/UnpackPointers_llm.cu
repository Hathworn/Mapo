#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;

    // Loop restructuring and use of shared memory for efficient memory access
    for (int y = 0; y < h / 32; y++) {
        for (int x = tx; x < w; x += blockDim.x) {
            unsigned int val = minmax[y * w + x];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}