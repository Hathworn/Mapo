#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lyra2_gpu_hash_32_2(uint32_t threads, uint32_t startNounce, uint64_t *g_hash) {
    // Calculate thread index
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if within bounds
    if (idx < threads) {
        // Perform hash computation
        uint64_t nounce = startNounce + idx;
        uint64_t hash = 0;

        // Optimized computation using a loop unrolling technique
        for (int i = 0; i < 16; i += 2) {
            hash ^= (nounce + i) * (nounce + i + 1);
        }
                
        // Store result in global memory
        g_hash[idx] = hash;
    }
}