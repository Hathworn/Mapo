#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void atomicadaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w)
{
    // Improved iterator setup
    int xx = threadIdx.x + blockIdx.x * blockDim.x;
    int yy = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Compute offset
    int o = blockIdx.z; // Use 3D grid for batch processing
    int i = o;

    if (xx < output_w && yy < output_h) { // Boundary check
        // Select input/output plane
        gradOutput += o * output_w * output_h;
        gradInput += i * input_w * input_h;
        indices_x += o * output_w * output_h;
        indices_y += o * output_w * output_h;

        // Compute input coordinates
        int y_start = (yy * input_h) / output_h;
        int x_start = (xx * input_w) / output_w;

        float *ptr_gradInput = gradInput + y_start * input_w + x_start;
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float *ptr_ind_x = indices_x + yy * output_w + xx;
        float *ptr_ind_y = indices_y + yy * output_w + xx;
        float z = *ptr_gradOutput;

        int argmax_x = (*ptr_ind_x) - 1;
        int argmax_y = (*ptr_ind_y) - 1;

        // atomic add since different threads could update same variable
        atomicAdd(&(ptr_gradInput[argmax_x + argmax_y * input_w]), z);
    }
}