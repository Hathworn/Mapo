#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void max_gradInput(float *input, float *output, float *indices, long nrows, long ncols)
{
    // Calculate global thread index for output
    long o = blockIdx.x * blockDim.x + threadIdx.x;
    if (o < nrows) {
        // Compute input offset
        long i = o * ncols;
        
        // Index from indices array (convert from 1-based index)
        long idx = __ldg(&indices[o]) - 1;

        // Optimized memory access
        input[i + idx] = output[o];
    }
}