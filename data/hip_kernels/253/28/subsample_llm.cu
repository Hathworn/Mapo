#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx = threadIdx.x;
    int yy = blockDim.y * blockIdx.y + threadIdx.y;

    // Select input/output plane
    output += o * output_w * output_h;
    input += i * input_w * input_h;

    // Check bounds to ensure threads operate within the output limits
    if (yy < output_h) {
        for (; xx < output_w; xx += blockDim.x) {
            // Compute the mean of the input image
            float sum = 0.0f;
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    sum += input[(yy * dH + ky) * input_w + xx * dW + kx];
                }
            }
            // Update output
            output[yy * output_w + xx] = sum / (kW * kH);
        }
    }
}