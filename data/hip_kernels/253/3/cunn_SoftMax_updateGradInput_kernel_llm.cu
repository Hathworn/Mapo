#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_SoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int nframe, int dim)
{
    // Use warp shuffle to implement reduction for better performance
    __shared__ float buffer[SOFTMAX_THREADS];
    int k = blockIdx.x;
    float *gradInput_k = gradInput + k * dim;
    float *output_k = output + k * dim;
    float *gradOutput_k = gradOutput + k * dim;

    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;

    float threadSum = 0;
    for (int i = i_start; i < i_end; i += i_step)
        threadSum += gradOutput_k[i] * output_k[i];

    // Perform reduction using warp shuffle
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
        threadSum += __shfl_down(threadSum, offset);

    // Write result from each warp back to shared memory
    if (threadIdx.x % warpSize == 0)
        buffer[threadIdx.x / warpSize] = threadSum;

    __syncthreads();

    // Final reduction among warps
    float sum_k = 0;
    if (threadIdx.x < blockDim.x / warpSize) {
        sum_k = buffer[threadIdx.x];
        for (int offset = blockDim.x / warpSize / 2; offset > 0; offset /= 2)
            sum_k += __shfl_down(sum_k, offset);

        if(threadIdx.x == 0)
            buffer[0] = sum_k;
    }

    __syncthreads();

    sum_k = buffer[0];
    for (int i = i_start; i < i_end; i += i_step)
        gradInput_k[i] = output_k[i] * (gradOutput_k[i] - sum_k);
}