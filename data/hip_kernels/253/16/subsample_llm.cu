#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Compute thread indexes
    int xx = threadIdx.x + blockDim.x * blockIdx.x;
    int yy = threadIdx.y + blockDim.y * blockIdx.y;

    // Output size calculation
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Return if thread is out of output bounds
    if (xx >= output_w || yy >= output_h) return;

    // Compute offsets based on thread/block ID
    int o = blockIdx.z;
    int k = o % input_n;
    output += o * output_w * output_h;
    input += o * input_w * input_h;

    // Select input/output plane
    float the_weight = weight[k];
    float the_bias = bias[k];

    // Calculate the sum of the input image using a coalesced access pattern
    float sum = 0;
    input += yy * dH * input_w + xx * dW;
    for (int ky = 0; ky < kH; ++ky) {
        for (int kx = 0; kx < kW; ++kx) {
            sum += input[kx];
        }
        input += input_w; // move to next input line
    }

    // Write calculated value to output with weight and bias applied
    output[yy * output_w + xx] = the_weight * sum + the_bias;
}