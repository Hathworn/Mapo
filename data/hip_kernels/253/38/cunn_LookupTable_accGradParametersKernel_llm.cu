#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_LookupTable_accGradParametersKernel( float *input, float *indices, float *gradOutput, float *gradWeight, float *count, float defaultScale, long numel, long stride) {

    int idx = blockIdx.x * 4 + threadIdx.y;

    // Precompute values used multiple times in loop
    const int SZ = 4;
    const int warpStartFeature = blockIdx.y * blockDim.x;

    // Process if idx is in range and it's the start of a new sequence
    if (idx < numel && (idx == 0 || input[idx] != input[idx - 1])) {
        do {
            const int startFeature = threadIdx.x + warpStartFeature;
            const int weightRow = ((int) input[idx] - 1) * stride;
            const int gradOutputRow = ((int) indices[idx] - 1) * stride;
            const float scale = count ? defaultScale / count[idx] : defaultScale;

            float gradient[SZ] = {0.0};
            float weight[SZ] = {0.0};

            // Unroll and optimize memory accesses
            #pragma unroll
            for (int ii = 0; ii < SZ; ii++) {
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride) {
                    gradient[ii] = gradOutput[gradOutputRow + featureDim];
                    weight[ii] = gradWeight[weightRow + featureDim];
                }
            }

            // Parallelize gradient update
            #pragma unroll
            for (int ii = 0; ii < SZ; ii++) {
                weight[ii] += gradient[ii] * scale;
            }

            // Efficient memory write-back
            #pragma unroll
            for (int ii = 0; ii < SZ; ii++) {
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride) {
                    gradWeight[weightRow + featureDim] = weight[ii];
                }
            }

            idx++;
        } while (idx < numel && input[idx] == input[idx - 1]);
    }
}