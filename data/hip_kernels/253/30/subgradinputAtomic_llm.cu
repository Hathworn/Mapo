#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;

    int yy_start = blockIdx.y * blockDim.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;

    // Optimize memory access and computation
    float scale = 1.0f / (kW * kH);

    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float z = gradOutput[yy * output_w + xx];
            for (int ky = 0; ky < kH; ++ky) {
                for (int kx = 0; kx < kW; ++kx) {
                    // Optimize atomic operation
                    atomicAdd(&ptr_gradInput[kx], z * scale);
                }
                ptr_gradInput += input_w;
            }
        }
    }
}