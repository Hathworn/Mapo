#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate the unique index for each thread based on block and thread IDs
    int global_tid = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    int batch_offset = blockIdx.x * input_w * input_n;

    // Set pointers for gradInput, gradOutput, and indices
    float *gradInput_data = gradInput + batch_offset + global_tid * input_n * dW;
    float *gradOutput_data = gradOutput + batch_offset + global_tid * input_n;
    float *indices_data = indices + batch_offset + global_tid * input_n;

    // Ensure the current thread is processing a valid output width
    if (global_tid < output_w) {
        // Loop over all features
        for (int feat = 0; feat < input_n; ++feat) {
            // Atomic add for safe updates to gradInput
            atomicAdd(&gradInput_data[(int)indices_data[feat] * input_n + feat], gradOutput_data[feat]);
        }
    }
}