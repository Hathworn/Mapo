#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_MSECriterion_updateOutput_kernel(float* output, float *input, float *target, int nframe, int dim, int sizeAverage)
{
    __shared__ float buffer[MSECRITERION_THREADS];
    int k = blockIdx.x;
    float *input_k = input + k*dim;
    float *target_k = target + k*dim;
    
    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;

    // Initialize shared memory buffer
    buffer[threadIdx.x] = 0;

    // Efficient overlap of computation by each thread
    for (int i = i_start; i < i_end; i += i_step)
    {
        float z = input_k[i] - target_k[i];
        buffer[threadIdx.x] += z * z;
    }
    __syncthreads();

    // Parallel reduction for buffer elements
    int tid = threadIdx.x;
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if (tid < offset) // Avoid race conditions
        {
            buffer[tid] += buffer[tid + offset];
        }
        __syncthreads();
    }

    // Final write-back by the first thread
    if (tid == 0)
    {
        *output = buffer[0];
        if (sizeAverage)
            *output /= dim;
    }
}