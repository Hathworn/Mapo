#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Block idx is the batch index, thread idx + block idx y * MAX_THREADS is the time index
    int timeIdx = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    if (timeIdx < output_w) {  // Early termination check
        float *gradInput_data = gradInput + blockIdx.x * input_w * input_n + timeIdx * input_n * dW;
        float *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + timeIdx * input_n;
        float *indices_data = indices + blockIdx.x * output_w * input_n + timeIdx * input_n;

        // Unroll loop (assuming input_n multiple of 4 for demonstration, adjust as needed)
        for (int feat = 0; feat < input_n; feat += 4) {
            int idx0 = (int)indices_data[feat] * input_n + feat;
            int idx1 = (int)indices_data[feat + 1] * input_n + feat + 1;
            int idx2 = (int)indices_data[feat + 2] * input_n + feat + 2;
            int idx3 = (int)indices_data[feat + 3] * input_n + feat + 3;
            
            gradInput_data[idx0] += gradOutput_data[feat];
            gradInput_data[idx1] += gradOutput_data[feat + 1];
            gradInput_data[idx2] += gradOutput_data[feat + 2];
            gradInput_data[idx3] += gradOutput_data[feat + 3];
        }
    }
}