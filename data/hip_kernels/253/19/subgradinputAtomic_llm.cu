#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on block index and thread index
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_step = blockDim.x;
    int yy_step = blockDim.y * gridDim.y;

    // Start positions for threads
    int xx_start = threadIdx.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;

    // Adjust pointers based on current block
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;

    // Get the weight for the current index
    float the_weight = weight[k];

    // Compute gradInput
    for(int yy = yy_start; yy < output_h; yy += yy_step) {
        for(int xx = xx_start; xx < output_w; xx += xx_step) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput * the_weight;
            
            // Use shared memory for partial results 
            for(int ky = 0; ky < kH; ky++) {
                for(int kx = 0; kx < kW; kx++) {
                    atomicAdd(&(ptr_gradInput[kx]), z);
                }
                ptr_gradInput += input_w; // Move to next row
            }
        }
    }
}
```
