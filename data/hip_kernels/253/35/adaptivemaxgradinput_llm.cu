#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w)
{
    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // Precompute scaling factors
    float scale_h = float(input_h) / output_h;
    float scale_w = float(input_w) / output_w;

    // Parallelize over output dimensions using a single loop
    for (int index = yy_start * output_w + xx_start; index < output_h * output_w; index += yy_step * output_w + xx_step) {
        int xx = index % output_w;
        int yy = index / output_w;

        int y_start = (int)floor(yy * scale_h);
        int x_start = (int)floor(xx * scale_w);

        float *ptr_gradInput = gradInput + y_start * input_w + x_start;
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float *ptr_ind_x = indices_x + yy * output_w + xx;
        float *ptr_ind_y = indices_y + yy * output_w + xx;
        float z = *ptr_gradOutput;

        int argmax_x = (*ptr_ind_x) - 1;
        int argmax_y = (*ptr_ind_y) - 1;

        atomicAdd(&ptr_gradInput[argmax_x + argmax_y * input_w], z);  // Use atomicAdd for potential race conditions
    }
}