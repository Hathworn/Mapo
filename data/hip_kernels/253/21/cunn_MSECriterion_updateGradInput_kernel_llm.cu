#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_MSECriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, float norm, int nframe, int dim)
{
    // Calculate flattened global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread index is within bounds
    if (index < nframe * dim) {
        // Calculate indices for the respective arrays
        int k = index / dim;
        int i = index % dim;

        // Calculate pointers for offsets
        float *gradInput_k = gradInput + k * dim;
        float *input_k = input + k * dim;
        float *target_k = target + k * dim;

        // Compute gradient
        gradInput_k[i] = norm * (input_k[i] - target_k[i]);
    }
}