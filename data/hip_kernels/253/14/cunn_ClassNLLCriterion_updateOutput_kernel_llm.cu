#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_ClassNLLCriterion_updateOutput_kernel(float *output, float *input, float *target, int nframe, int ndim, int sizeAverage, int ntarget) {
    extern __shared__ float shInputs[];
    int tid = threadIdx.x;
    int index;
    float localSum = 0.0f;

    // Calculate partial sum for each thread
    for (int i = tid; i < nframe; i += blockDim.x) {
        for (int j = 0; j < ntarget; ++j) {
            index = static_cast<int>(target[i * ntarget + j]) - 1;
            if (index >= 0) {
                localSum += input[i * ndim + index];
            }
        }
    }
    shInputs[tid] = localSum;
    __syncthreads();

    // Reduce partial sums to a single value
    if (tid == 0) {
        float totalSum = 0.0f;
        for (int i = 0; i < blockDim.x; ++i) {
            totalSum += shInputs[i];
        }
        if (sizeAverage) {
            totalSum /= nframe;
        }
        *output = -totalSum;
    }
}