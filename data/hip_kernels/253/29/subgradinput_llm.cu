#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinput(float *gradInput, float *gradOutput, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Offset pointers for current output/input features
    gradOutput = gradOutput + o * output_w * output_h;
    gradInput = gradInput + i * input_w * input_h;

    // Calculate gradient input
    for(int yy = yy_start; yy < output_h; yy += yy_step) {
        for(int xx = xx_start; xx < output_w; xx += xx_step) {
            // Pointer calculations outside inner loops
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput;
            float increment = z / float(kW * kH);
            
            // Unroll inner loops for better performance
            for(int ky = 0; ky < kH; ky++) {
                for(int kx = 0; kx < kW; kx++) {
                    ptr_gradInput[kx] += increment;
                }
                ptr_gradInput += input_w;
            }
        }
    }
}