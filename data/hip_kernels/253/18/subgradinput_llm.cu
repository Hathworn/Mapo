#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinput(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int k = o % input_n;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;

    int yy_start = blockIdx.y * blockDim.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += o * input_w * input_h;

    // Get weight
    float the_weight = weight[k];

    // Compute gradInput
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float z = gradOutput[yy * output_w + xx] * the_weight; // Direct access to gradOutput

            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    ptr_gradInput[kx] += z; // Accumulate the product
                }
                ptr_gradInput += input_w; // Move to the next row
            }
        }
    }
}