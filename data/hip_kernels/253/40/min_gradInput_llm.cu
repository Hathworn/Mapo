#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void min_gradInput(float *input, float *output, float *indices, long nrows, long ncols)
{
    // Calculate the overall index for this thread
    long o = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if the current index is within bounds
    if (o < nrows) {
        // Calculate input offset
        long i = o * ncols;
        
        // Get the index from the indices array and adjust for 0-based indexing
        long idx = static_cast<long>(indices[o]) - 1;
        
        // Assign the output value to the corresponding position in the input
        input[i + idx] = output[o];
    }
}