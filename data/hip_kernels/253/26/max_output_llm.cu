#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void max_output(float *input, float *output, float *indices, long nrows, long ncols)
{
    // Calculate output offset
    long o = threadIdx.x + blockDim.x * blockIdx.x;
    if (o >= nrows) return;

    // Calculate input offset
    long i = o * ncols;

    // Initialize max and argmax
    float max = input[i];
    long argmax = 0;

    // Iterate over columns to find the max value and its index
    for (long ii = 1; ii < ncols; ++ii) {
        float val = input[i + ii];
        if (val > max) {
            max = val;
            argmax = ii;
        }
    }

    // Store max value and 1-based index
    output[o] = max;
    indices[o] = argmax + 1;
}