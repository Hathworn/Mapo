#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void maxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // optimizes block and thread computation for efficiency
    int o = blockIdx.x;
    
    // precompute and cache values to reduce redundant computation
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;
    int xx_start = threadIdx.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int xx_step = blockDim.x;
    int yy_step = blockDim.y * gridDim.y;

    // adjust pointers for the current plane
    gradOutput += o * output_w * output_h;
    gradInput += o * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // loop through output dimensions efficiently
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float *ptr_ind_x = indices_x + yy * output_w + xx;
            float *ptr_ind_y = indices_y + yy * output_w + xx;
            float z = *ptr_gradOutput;

            // optimized index calculation and addition
            int argmax_x = (*ptr_ind_x) - 1;
            int argmax_y = (*ptr_ind_y) - 1;
            atomicAdd(&ptr_gradInput[argmax_x + argmax_y * input_w], z);
        }
    }
}