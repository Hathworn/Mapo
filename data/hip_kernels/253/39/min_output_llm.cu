#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void min_output(float *input, float *output, float *indices, long nrows, long ncols)
{
    // Output offset:
    long o = threadIdx.x + blockDim.x * blockIdx.x;
    if (o >= nrows) return;

    // Input offset:
    long i = o * ncols;

    // Move pointers
    input = input + i;

    // Initialize min and argmin variables
    float min = input[0];
    long argmin = 0;

    // Compute min using loop unrolling
    #pragma unroll 4
    for (long ii = 1; ii < ncols; ii++) {
        float val = input[ii];
        if (val < min) {
            min = val;
            argmin = ii;
        }
    }

    // Store results
    output[o] = min;
    indices[o] = argmin + 1;
}