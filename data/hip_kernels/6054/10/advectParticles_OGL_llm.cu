#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void advectParticles_OGL(float2 *part, float2 *v, int dx, int dy, float dt, int lb, size_t pitch) {
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;
    
    float2 pterm, vterm;

    // Check boundaries only once for efficiency
    if (gtidx < dx) {
        for (p = 0; p < lb && gtidy + p < dy; ++p) {
            int fi = gtidy + p;
            int fj = fi * dx + gtidx;

            // Direct access float2 array for particles using linear indexing
            pterm = part[fj];

            // Calculate velocity index accurately
            int xvi = min(max((int)(pterm.x * dx), 0), dx - 1);
            int yvi = min(max((int)(pterm.y * dy), 0), dy - 1);

            // Simplified velocity access
            vterm = *((float2*)((char*)v + yvi * pitch) + xvi);

            // Efficient periodic boundary condition application
            pterm.x = fmodf(pterm.x + dt * vterm.x + 1.f, 1.f);
            pterm.y = fmodf(pterm.y + dt * vterm.y + 1.f, 1.f);

            part[fj] = pterm;
        }
    }
}