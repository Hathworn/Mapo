#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square(int *array, int arrayCount)
{
    extern __shared__ int dynamicSmem[];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;  // Calculate stride for grid-stride loop

    // Grid-stride loop to utilize more threads and improve performance
    for (int i = idx; i < arrayCount; i += stride) {
        array[i] *= array[i];
    }
}