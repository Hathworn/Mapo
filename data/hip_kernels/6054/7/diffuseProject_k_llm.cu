#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diffuseProject_k(float2 *vx, float2 *vy, int dx, int dy, float dt, float visc, int lb) {
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;

    // Unroll loops to increase parallelism and reduce overhead
    for (int p = 0; p < lb; p++) {
        int fi = gtidy + p;
        if (gtidx < dx && fi < dy) { // Combine condition checks to reduce branching
            int fj = fi * dx + gtidx;
            float2 xterm = vx[fj];
            float2 yterm = vy[fj];

            int iix = gtidx;
            int iiy = (fi > dy / 2) ? (fi - dy) : fi;

            // Pre-calculate values to reuse them
            float kk = (float)(iix * iix + iiy * iiy);
            float diff = 1.f / (1.f + visc * dt * kk);
            xterm.x *= diff;
            xterm.y *= diff;
            yterm.x *= diff;
            yterm.y *= diff;

            if (kk > 0.f) {
                float rkk = 1.f / kk;
                float rkp = (iix * xterm.x + iiy * yterm.x);
                float ikp = (iix * xterm.y + iiy * yterm.y);
                float common_x = rkk * iix;
                float common_y = rkk * iiy;

                // Reduce redundant calculations
                xterm.x -= common_x * rkp;
                xterm.y -= common_x * ikp;
                yterm.x -= common_y * rkp;
                yterm.y -= common_y * ikp;
            }

            vx[fj] = xterm;
            vy[fj] = yterm;
        }
    }
}