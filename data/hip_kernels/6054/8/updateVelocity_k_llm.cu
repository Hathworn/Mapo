#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateVelocity_k(float2 *v, float *vx, float *vy, int dx, int pdx, int dy, int lb, size_t pitch) {
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int baseY = blockIdx.y * (lb * blockDim.y);
    float scale = 1.f / (dx * dy); // Precompute scale factor
    if (gtidx < dx) {
        for (int p = 0; p < lb; p++) {
            int fi = baseY + threadIdx.y * lb + p;
            if (fi < dy) {
                int fjr = fi * pdx + gtidx;
                float vxterm = vx[fjr]; // Directly use inline variables
                float vyterm = vy[fjr];
                
                float2 nvterm;
                nvterm.x = vxterm * scale;
                nvterm.y = vyterm * scale;

                // Simplify memory access for float2* fj
                float2 *fj = (float2*)((char*)v + fi * pitch) + gtidx;
                *fj = nvterm;
            }
        }
    }
}