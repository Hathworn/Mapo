#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Utilize registers and shared memory to improve performance
__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int local_factor = *factor;  // Use register to store factor

    // Unrolling loop to minimize loop overhead
    for (int i = 0; i < num_iterations; i += 4) {
        g_data[idx] += local_factor;
        g_data[idx] += local_factor;
        g_data[idx] += local_factor;
        g_data[idx] += local_factor;
    }

    // Handle remaining iterations for non-multiples of 4
    for (int i = num_iterations & ~3; i < num_iterations; ++i) {
        g_data[idx] += local_factor;
    }
}