#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addForces_k(float2 *v, int dx, int dy, int spx, int spy, float fx, float fy, int r, size_t pitch) {
    // Calculate the unique thread index for memory access
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // Pointer to the specific element in the 2D v array
    float2 *fj = (float2*)((char*)v + (ty + spy) * pitch) + tx + spx;

    // Load the value at the fj position into the register
    float2 vterm = *fj;

    // Adjust 'tx' and 'ty' with radius 'r'
    int adjusted_tx = tx - r;
    int adjusted_ty = ty - r;
    
    // Pre-compute powers to improve performance
    float tx4 = adjusted_tx * adjusted_tx * adjusted_tx * adjusted_tx;
    float ty4 = adjusted_ty * adjusted_ty * adjusted_ty * adjusted_ty;
    
    // Compute scaling factor 's'
    float s = 1.f / (1.f + tx4 + ty4);
    
    // Update the x and y components of vterm
    vterm.x += s * fx;
    vterm.y += s * fy;

    // Write the updated value back to global memory
    *fj = vterm;
}
```
