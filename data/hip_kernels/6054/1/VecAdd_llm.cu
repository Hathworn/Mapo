#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAdd(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int N)
{
    // Calculate global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Loop unrolling for better performance
    if (i < N)
    {
        C[i] = A[i] + B[i];
        if (i + blockDim.x < N) C[i + blockDim.x] = A[i + blockDim.x] + B[i + blockDim.x];
    }
}