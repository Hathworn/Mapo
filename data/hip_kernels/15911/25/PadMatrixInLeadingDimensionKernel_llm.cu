#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PadMatrixInLeadingDimensionKernel(const int8_t* src, int8_t* dst, int col_src, int col_dst) {
    int idx = blockIdx.x * col_src + threadIdx.x;
    int dst_idx = blockIdx.x * col_dst + threadIdx.x;
    // Loop unrolling for better memory coalescing
    for (int i = idx; i < col_src; i += blockDim.x * gridDim.x) {
        dst[dst_idx] = src[i];
    }
}