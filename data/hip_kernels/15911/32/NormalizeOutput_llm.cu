#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Calculate global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use while loop for coalesced memory access and reduce loop control overhead
    while (idx < num_elements) {
        int base_idx = idx * 3;  // Calculate base index for memory access
        to_normalize[base_idx] = batch_index;  // Set batch index
        to_normalize[base_idx + 1] = class_index;  // Set class index
        to_normalize[base_idx + 2] = static_cast<int64_t>(original[idx]);  // Set original value
        idx += blockDim.x * gridDim.x;  // Move to next set of data
    }
}