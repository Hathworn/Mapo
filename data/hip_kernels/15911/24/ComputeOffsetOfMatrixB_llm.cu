#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Calculate index using block and thread id for coalesced memory access
    int32_t idx = blockIdx.x * N + threadIdx.x;
    if (threadIdx.x < N) {
        // Assign offset computation directly using calculated index
        output[idx] = -row_sum[blockIdx.x];
    }
}