#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    int32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    int32_t stride = blockDim.x * gridDim.x;
    // Optimize loop to handle larger data sets by iterating with stride
    for (int32_t i = idx; i < N; i += stride) {
        *(output + blockIdx.x * N + i) = -col_sum[i];
    }
}