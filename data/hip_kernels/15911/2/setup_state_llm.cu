#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Optimize by using threadIdx.x to initialize each state for concurrent execution.
    hiprand_init(seed, threadIdx.x, 0, &state[threadIdx.x]);
}