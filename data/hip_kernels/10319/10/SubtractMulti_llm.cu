#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    const int x = blockIdx.x * SUBTRACTM_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACTM_H + threadIdx.y;
    const int z = threadIdx.z; // Cache threadIdx.z to avoid accessing the register multiple times
    int sz = height * pitch;
    int p = z * sz + y * pitch + x;
    
    if (x < width && y < height) {
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
    // __syncthreads() removed as it is unnecessary here; no shared memory dependency
}