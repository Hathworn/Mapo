#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;
    // Optimize loop bounds and memory access pattern
    for (int y = 0; y < h; y += 32) {
        for (int x = 0; x < w; x += 16) {
            unsigned int val = minmax[(y/32)*w + x + tx];
            while (val && numPts < maxPts) {
                // Find the position of the next set bit (if any)
                int k = __ffs(val) - 1;
                if (k != -1 && numPts < maxPts) {
                    ptrs[16 * numPts + tx] = (y + k) * w + x + tx;
                    numPts++;
                }
                // Reset the found bit
                val &= ~(1U << k);
            }
        }
    }
}