#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOrientations(float *g_Data, float *d_Sift, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Initialize histogram to zero
    for (int i = tx; i < 13 * 32; i += 32)
        hist[i] = 0.0f;
    __syncthreads();

    // Compute precomputed gaussian values
    if (tx < 15)
        gauss[tx] = exp(-1.0f / (2.0f * 9.0f) * (tx - 7) * (tx - 7));

    int xp = (int)(d_Sift[bx + 0 * maxPts] - 6.5f);
    int yp = (int)(d_Sift[bx + 1 * maxPts] - 6.5f);
    int px = xp & 15;
    int x = tx - px;

    // Load data into shared memory
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = max(0, min(w - 1, xp + x));
        int yi = max(0, min(h - 1, yp + y));
        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    // Compute gradients and populate histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = (int)(16.0f * atan2f(dy, dx) / 3.1416f) & 31;
            float grad = sqrtf(dx * dx + dy * dy);
            hist[32 * (x - 1) + bin] += grad * gauss[x] * gauss[y];
        }
    }
    __syncthreads();

    // Reduce along y-axis
    for (int offset = 8; offset > 0; offset >>= 1) {
        for (int y = tx; y < offset * 32; y += 32) {
            hist[y] += hist[(y + offset * 32)];
        }
        __syncthreads();
    }
    
    // Post-process histogram
    float v = hist[tx];
    hist[tx] = (v > 0 && v > hist[(tx + 1) & 31] && v >= hist[(tx + 31) & 31]) ? v : 0.0f;
    __syncthreads();

    // Identify peaks
    if (tx == 0) {
        float maxval1 = 0.0, maxval2 = 0.0;
        int i1 = -1, i2 = -1;
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1; maxval1 = v; 
                i2 = i1; i1 = i;
            } else if (v > maxval2) {
                maxval2 = v; i2 = i;
            }
        }

        // Store dominant orientations
        float val1 = hist[(i1 + 1) & 31];
        float val2 = hist[(i1 + 31) & 31];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Sift[bx + 5 * maxPts] = 11.25f * ((peak < 0.0f) ? (peak + 32.0f) : peak);

        if (maxval2 < 0.8f * maxval1) i2 = -1;
        if (i2 >= 0) {
            val1 = hist[(i2 + 1) & 31];
            val2 = hist[(i2 + 31) & 31];
            peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Sift[bx + 6 * maxPts] = 11.25f * ((peak < 0.0f) ? (peak + 32.0f) : peak);
        } else {
            d_Sift[bx + 6 * maxPts] = i2;
        }
    }
}