#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;  // Replace deprecated __mul24 with regular multiplication
    const int y = blockIdx.y * 16;
    int p = y * width + x;
    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        minvals[tx] = maxvals[tx] = FLT_MAX;  // Initiate with neutral values for reduction
    }
    __syncthreads();

    for (int ty = 1; ty < 16; ty++) {
        p += width;
        if (x < width) {
            float val = d_Data[p];
            if (val < minvals[tx])
                minvals[tx] = val;
            if (val > maxvals[tx])
                maxvals[tx] = val;
        }
    }
    __syncthreads();

    for (int stride = 64; stride > 0; stride >>= 1) {  // Use stride-based reduction
        if (tx < stride) {
            minvals[tx] = fminf(minvals[tx], minvals[tx + stride]);
            maxvals[tx] = fmaxf(maxvals[tx], maxvals[tx + stride]);
        }
        __syncthreads();
    }

    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}