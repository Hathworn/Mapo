#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define POSBLK_SIZE 256

__global__ void ComputePositions(float *g_Data1, float *g_Data2, float *g_Data3, int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim.x for flexibility
    if (i >= numPts)
        return;

    int p = d_Ptrs[i];

    // Preload data to registers
    float val0 = g_Data2[p];
    float val1 = g_Data2[p-1];
    float val2 = g_Data2[p+1];
    float val3 = g_Data2[p-w];
    float val4 = g_Data2[p+w];
    float val5 = g_Data3[p];
    float val6 = g_Data1[p];
    
    // Compute derivatives
    float dx = 0.5f * (val2 - val1);
    float dxx = 2.0f * val0 - val1 - val2;
    float dy = 0.5f * (val4 - val3);
    float dyy = 2.0f * val0 - val3 - val4;
    float ds = 0.5f * (val6 - val5);
    float dss = 2.0f * val0 - val5 - val6;
    
    float dxy = 0.25f * (g_Data2[p+w+1] + g_Data2[p-w-1] - g_Data2[p-w+1] - g_Data2[p+w-1]);
    float dxs = 0.25f * (g_Data3[p+1] + g_Data1[p-1] - g_Data1[p+1] - g_Data3[p-1]);
    float dys = 0.25f * (g_Data3[p+w] + g_Data1[p-w] - g_Data3[p-w] - g_Data1[p+w]);

    // Compute inverse of Hessian matrix and determinant
    float idxx = dyy * dss - dys * dys;
    float idxy = dys * dxs - dxy * dss;
    float idxs = dxy * dys - dyy * dxs;
    float idyy = dxx * dss - dxs * dxs;
    float idys = dxy * dxs - dxx * dys;
    float idss = dxx * dyy - dxy * dxy;
    float det = idxx * dxx + idxy * dxy + idxs * dxs;
    float idet = 1.0f / det;

    // Position adjustment
    float pdx = idet * (idxx * dx + idxy * dy + idxs * ds);
    float pdy = idet * (idxy * dx + idyy * dy + idys * ds);
    float pds = idet * (idxs * dx + idys * dy + idss * ds);

    if (pdx < -0.5f || pdx > 0.5f || pdy < -0.5f || pdy > 0.5f || pds < -0.5f || pds > 0.5f) {
        pdx = __fdividef(dx, dxx);
        pdy = __fdividef(dy, dyy);
        pds = __fdividef(ds, dss);
    }

    float dval = 0.5f * (dx * pdx + dy * pdy + ds * pds);

    // Write results to global memory
    d_Sift[i + 0 * maxPts] = (p % w) + pdx;
    d_Sift[i + 1 * maxPts] = (p / w) + pdy;
    d_Sift[i + 2 * maxPts] = d_ConstantA[0] * exp2f(pds * d_ConstantB[0]);
    d_Sift[i + 3 * maxPts] = val0 + dval;

    // Compute trace and determinant for contrast value
    float tra = dxx + dyy;
    det = dxx * dyy - dxy * dxy;
    d_Sift[i + 4 * maxPts] = __fdividef(tra * tra, det);
}