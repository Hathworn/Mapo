#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LowPassColMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    __shared__ float data[CONVCOL_W*(CONVCOL_H + 2*RADIUS)];
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int block = blockIdx.x/(NUM_SCALES+3);
    const int scale = blockIdx.x - (NUM_SCALES+3)*block;
    const int miny = blockIdx.y*CONVCOL_H;
    const int maxy = min(miny + CONVCOL_H, height) - 1;
    const int totStart = miny - RADIUS;
    const int totEnd = maxy + RADIUS;
    const int colStart = block*CONVCOL_W + tx;
    const int sStep = CONVCOL_W*CONVCOL_S;
    const int gStep = pitch*CONVCOL_S;
    float *kernel = d_Kernel + scale*16;
    const int size = pitch*height*scale;

    d_Result += size;
    d_Data += size;

    if (colStart < width) {
        float *sdata = data + ty*CONVCOL_W + tx;
        int gPos = colStart + (totStart + ty) * pitch;

        for (int y = totStart + ty; y <= totEnd; y += blockDim.y) {
            if (y < 0)
                sdata[0] = d_Data[colStart];  // Out of bounds check
            else if (y >= height)
                sdata[0] = d_Data[colStart + (height - 1) * pitch];  // Out of bounds check
            else
                sdata[0] = d_Data[gPos];

            sdata += sStep;
            gPos += gStep;
        }
    }
    __syncthreads();

    if (colStart < width) {
        float *sdata = data + ty*CONVCOL_W + tx;
        int gPos = colStart + (miny + ty) * pitch;

        for (int y = miny + ty; y <= maxy; y += blockDim.y) {
            float result = 0.0f;
            for (int k = 0; k < 5; ++k) {
                result += (sdata[k*CONVCOL_W] + sdata[(8-k)*CONVCOL_W]) * kernel[k];  // Unrolling and symmetric access
            }
            result += sdata[4*CONVCOL_W] * kernel[4]; // Add center pixel contribution only once
            d_Result[gPos] = result;

            sdata += sStep;
            gPos += gStep;
        }
    }
}