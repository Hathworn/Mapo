#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Simplified calculation for x
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Simplified calculation for y
    int p = y * width + x; // Simplified index calculation
    if (x < width && y < height) // Check boundary conditions
    {
        d_Result[p] = d_Data1[p] - d_Data2[p]; // Perform subtraction
    }
    // __syncthreads() removed as it is unnecessary here
}