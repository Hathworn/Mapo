#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];
    
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    
    // Initialize histogram to zero
    for (int i = 0; i < 13; i++) {
        hist[i*32+tx] = 0.0f;
    }
    __syncthreads();
    
    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    
    // Precompute Gaussian values
    if (tx < 16) {
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));
    }
    
    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;
    int px = xp & 15;
    int x = tx - px;

    // Load 15x15 neighborhood data
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = max(0, min(w - 1, xp + x));
        int yi = max(0, min(h - 1, yp + y));
        if (x >= 0 && x < 15) {
            data[memPos] = g_Data[yi * w + xi];
        }
    }
    __syncthreads();

    // Compute gradients and histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1]  - data[memPos - 1];
            int bin = __float2int_rz(16.0f * atan2f(dy, dx) / 3.1416f + 16.5f) % 32;
            float grad = sqrtf(dx * dx + dy * dy);
            atomicAdd(&hist[32 * (x - 1) + bin], grad * gauss[x] * gauss[y]);
        }
    }
    __syncthreads();

    // Reduce histograms
    for (int offset = 8; offset > 0; offset /= 2) {
        for (int y = 0; y < offset; y++) {
            hist[y * 32 + tx] += hist[(y + offset) * 32 + tx];
        }
        __syncthreads();
    }

    // Weighted peak calculation
    if (tx < 32) {
        float histVal = hist[tx] + hist[32 + tx];
        hist[32 + tx] = (histVal > hist[32 + ((tx + 1) % 32)] && histVal >= hist[32 + ((tx + 31) % 32)]) ? histVal : 0.0f;
    }
    __syncthreads();

    // Find peaks
    if (tx == 0) {
        float maxval1 = 0.0f;
        float maxval2 = 0.0f;
        int i1 = -1;
        int i2 = -1;

        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }

        float val1 = hist[32 + ((i1 + 1) % 32)];
        float val2 = hist[32 + ((i1 + 31) % 32)];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Orient[bx] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);

        if (maxval2 >= 0.8f * maxval1 && i2 >= 0) {
            val1 = hist[32 + ((i2 + 1) % 32)];
            val2 = hist[32 + ((i2 + 31) % 32)];
            peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Orient[bx + maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        } else {
            d_Orient[bx + maxPts] = i2;
        }
    }
}