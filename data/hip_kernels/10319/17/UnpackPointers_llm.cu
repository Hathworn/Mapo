#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;

    // Precompute reused values.
    int txOffset = tx + blockIdx.x * blockDim.x;
    
    for (int y = 0; y < h / 32; y++) {
        for (int x = 0; x < w; x += 16) {
            unsigned int val = minmax[y * w + x + txOffset];
            
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + txOffset] = (y * 32 + k) * w + x + txOffset;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}