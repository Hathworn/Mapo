#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuSearchDoublet( const int* nSpM, const float* spMmat, const int* nSpB, const float* spBmat, const int* nSpT, const float* spTmat, const float* deltaRMin, const float* deltaRMax, const float* cotThetaMax, const float* collisionRegionMin, const float* collisionRegionMax, int* nSpMcomp, int* nSpBcompPerSpM_Max, int* nSpTcompPerSpM_Max, int* nSpBcompPerSpM, int* nSpTcompPerSpM, int* McompIndex, int* BcompIndex, int* tmpBcompIndex, int* TcompIndex, int* tmpTcompIndex) {
    extern __shared__ float sharedMem[];
    int* mPos = (int*)sharedMem;
    int* isMcompat = (int*)&mPos[1];

    if (threadIdx.x == 0) {
        *isMcompat = false;
    }
    __syncthreads();

    float rM = spMmat[blockIdx.x + (*nSpM) * 3];
    float zM = spMmat[blockIdx.x + (*nSpM) * 2];

    int offset(0);

    // Combined loop for Bcomp and Tcomp
    while (offset < max(*nSpB, *nSpT)) {
        // Doublet search for bottom hits
        if (threadIdx.x + offset < *nSpB) {
            float rB = spBmat[threadIdx.x + offset + (*nSpB) * 3];
            float zB = spBmat[threadIdx.x + offset + (*nSpB) * 2];
            float deltaR = rM - rB;
            bool isBcompat = deltaR <= *deltaRMax && deltaR >= *deltaRMin;
            
            if (isBcompat) {
                float cotTheta = (zM - zB) / deltaR;
                float zOrigin = zM - rM * cotTheta;
                isBcompat = fabsf(cotTheta) <= *cotThetaMax && zOrigin >= *collisionRegionMin && zOrigin <= *collisionRegionMax;
                if (isBcompat) {
                    int bPos = atomicAdd(&nSpBcompPerSpM[blockIdx.x], 1);
                    tmpBcompIndex[bPos + (*nSpB) * blockIdx.x] = threadIdx.x + offset;
                }
            }
        }

        // Doublet search for top hits
        if (threadIdx.x + offset < *nSpT) {
            float rT = spTmat[threadIdx.x + offset + (*nSpT) * 3];
            float zT = spTmat[threadIdx.x + offset + (*nSpT) * 2];
            float deltaR = rT - rM;
            bool isTcompat = deltaR >= *deltaRMin && deltaR <= *deltaRMax;

            if (isTcompat) {
                float cotTheta = (zT - zM) / deltaR;
                float zOrigin = zM - rM * cotTheta;
                isTcompat = fabsf(cotTheta) <= *cotThetaMax && zOrigin >= *collisionRegionMin && zOrigin <= *collisionRegionMax;
                if (isTcompat) {
                    int tPos = atomicAdd(&nSpTcompPerSpM[blockIdx.x], 1);
                    tmpTcompIndex[tPos + (*nSpT) * blockIdx.x] = threadIdx.x + offset;
                }
            }
        }

        offset += blockDim.x;
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        if (nSpBcompPerSpM[blockIdx.x] > 0 && nSpTcompPerSpM[blockIdx.x] > 0) {
            *mPos = atomicAdd(nSpMcomp, 1);
            *isMcompat = true;
            McompIndex[*mPos] = blockIdx.x;

            atomicMax(nSpBcompPerSpM_Max, nSpBcompPerSpM[blockIdx.x]);
            atomicMax(nSpTcompPerSpM_Max, nSpTcompPerSpM[blockIdx.x]);
        }
    }

    __syncthreads();

    if (*isMcompat == true) {
        offset = 0;
        while (offset < max(nSpBcompPerSpM[blockIdx.x], nSpTcompPerSpM[blockIdx.x])) {
            if (threadIdx.x + offset < nSpBcompPerSpM[blockIdx.x]) {
                BcompIndex[threadIdx.x + offset + (*nSpB) * (*mPos)] = tmpBcompIndex[threadIdx.x + offset + (*nSpB) * blockIdx.x];
            }

            if (threadIdx.x + offset < nSpTcompPerSpM[blockIdx.x]) {
                TcompIndex[threadIdx.x + offset + (*nSpT) * (*mPos)] = tmpTcompIndex[threadIdx.x + offset + (*nSpT) * blockIdx.x];
            }
            offset += blockDim.x;
        }
    }
}