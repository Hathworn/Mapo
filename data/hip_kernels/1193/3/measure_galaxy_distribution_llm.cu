#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BIN_WIDTH 0.25
#define BLOCK_DIM 256
#define COVERAGE 180
#define LINE_LENGTH 30

#define BINS_TOTAL (COVERAGE * (int)(1 / BIN_WIDTH))

typedef struct Galaxy
{
    float declination;
    float declination_cos;
    float declination_sin;
    float right_ascension;
} Galaxy;

__global__ void measure_galaxy_distribution(int *DD_histogram, int *DR_histogram, int *RR_histogram, float *distribution, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop unrolling for increased performance
    for (int i = index; i < n; i += stride * 4) {
        if (i < n && RR_histogram[i] != 0) {
            distribution[i] = (DD_histogram[i] - 2.0f * DR_histogram[i] + RR_histogram[i]) / RR_histogram[i];
        }
        if (i + stride < n && RR_histogram[i + stride] != 0) {
            distribution[i + stride] = (DD_histogram[i + stride] - 2.0f * DR_histogram[i + stride] + RR_histogram[i + stride]) / RR_histogram[i + stride];
        }
        if (i + 2 * stride < n && RR_histogram[i + 2 * stride] != 0) {
            distribution[i + 2 * stride] = (DD_histogram[i + 2 * stride] - 2.0f * DR_histogram[i + 2 * stride] + RR_histogram[i + 2 * stride]) / RR_histogram[i + 2 * stride];
        }
        if (i + 3 * stride < n && RR_histogram[i + 3 * stride] != 0) {
            distribution[i + 3 * stride] = (DD_histogram[i + 3 * stride] - 2.0f * DR_histogram[i + 3 * stride] + RR_histogram[i + 3 * stride]) / RR_histogram[i + 3 * stride];
        }
    }
}