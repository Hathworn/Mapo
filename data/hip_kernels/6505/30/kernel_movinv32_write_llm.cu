#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);

    // Exit if pointer is outside bounds
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    unsigned int idx = threadIdx.x; // Use thread index for improved parallelism
    unsigned int pat = pattern;
    unsigned int k = (offset + idx) % 32; // Calculate initial k based on thread index

    // Calculate the pattern based on idx to avoid divergence
    for (unsigned int i = idx; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        ptr[i] = pat;
        k++;
        if (k >= 32) {
            k = 0;
            pat = lb;
        } else {
            pat = pat << 1;
            pat |= sval;
        }
    }

    return;
}