#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Early exit if pointer is out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Optimized loop using thread-level parallelism
    unsigned int tid = threadIdx.x;
    unsigned int stride = blockDim.x;
    for (unsigned int i = tid; i < BLOCKSIZE / sizeof(unsigned int); i += stride) {
        ptr[i] = pattern;
    }
}