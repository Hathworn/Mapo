#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    unsigned int* orig_ptr = reinterpret_cast<unsigned int*>(_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* ptr = orig_ptr;
    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);

    // Early exit if the initial pointer is outside the bounds.
    if (ptr >= reinterpret_cast<unsigned int*>(end_ptr)) {
        return;
    }

    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Write initial pattern.
    *ptr = pattern;

    // Optimized the loop to avoid unnecessary recalculations.
    while (true) {
        ptr = reinterpret_cast<unsigned int*>(((unsigned long)orig_ptr) | mask);
        if (ptr == orig_ptr || ptr >= block_end) {
            if (ptr >= block_end) break;
            mask <<= 1;
            continue;
        }

        *ptr = pattern;

        // Update pattern and mask.
        pattern <<= 1;
        mask <<= 1;
    }
}