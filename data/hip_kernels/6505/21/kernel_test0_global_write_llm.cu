#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;

    unsigned int pattern = 1;
    unsigned long mask = 4;

    *ptr = pattern;

    while (ptr < end_ptr) {
        ptr = (unsigned int*) ((((unsigned long)ptr) & ~(mask - 1)) | mask);  // Aligning pointer with mask
        if (ptr >= end_ptr) {
            break;  // Exit loop if pointer goes beyond end
        }
        *ptr = pattern;  // Write pattern to current pointer location
        pattern <<= 1;   // Shift pattern to the left
        mask <<= 1;      // Double the mask
    }
    return;
}