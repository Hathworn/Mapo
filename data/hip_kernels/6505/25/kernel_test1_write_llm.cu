#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err) {
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check boundary and exit early if out of range
    if (ptr >= (unsigned long*) end_ptr) {
        return;
    }

    // Calculate the loop limit once
    unsigned int limit = BLOCKSIZE / sizeof(unsigned long);

    // Unroll the loop for better performance
    for (unsigned int i = 0; i < limit; i += 4) {
        ptr[i] = (unsigned long) &ptr[i];
        ptr[i + 1] = (unsigned long) &ptr[i + 1];
        ptr[i + 2] = (unsigned long) &ptr[i + 2];
        ptr[i + 3] = (unsigned long) &ptr[i + 3];
    }

    return;
}