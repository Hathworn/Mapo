#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCKSIZE 256

__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Early exit if out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Calculate half_count once to optimize performance
    unsigned int half_count = BLOCKSIZE / sizeof(unsigned int) / 2;
    unsigned int* ptr_mid = ptr + half_count;

    // Use shared memory to optimize memory access patterns
    __shared__ unsigned int shared_data[BLOCKSIZE / sizeof(unsigned int)];

    // Load data into shared memory
    for (unsigned int i = threadIdx.x; i < half_count; i += blockDim.x) {
        shared_data[i] = ptr[i];
        shared_data[i + half_count] = ptr_mid[i];
    }

    __syncthreads();

    // Transfer data from shared memory to destination positions
    for (unsigned int i = threadIdx.x; i < half_count - 8; i += blockDim.x) {
        ptr[i + 8] = shared_data[half_count + i];
    }

    for (unsigned int i = threadIdx.x; i < 8; i += blockDim.x) {
        ptr[i] = shared_data[half_count - 8 + i];
    }
}