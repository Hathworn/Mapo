#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    // Calculate the global thread index
    unsigned int globalIdx = blockIdx.x * BLOCKSIZE / sizeof(unsigned int) + threadIdx.x;
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* start_ptr = (unsigned int*)_start_ptr;
    unsigned int* end_ptr_int = (unsigned int*)end_ptr;

    // Check if the current thread's work is within bounds
    if (globalIdx >= ((unsigned int*) end_ptr - ptr)) {
        return;
    }

    // Perform the memory copy operation
    ptr[globalIdx] = start_ptr[globalIdx];
}