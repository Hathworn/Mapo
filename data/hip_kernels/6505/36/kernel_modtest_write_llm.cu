#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCKSIZE 256 // Adjust according to your specific use-case
#define MOD_SZ 16     // Adjust according to your specific use-case

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Exit early if pointer exceeds end pointer
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Calculate valid index range and avoid repetitive division operations
    unsigned int num_elements = BLOCKSIZE / sizeof(unsigned int);
    
    // Write p1 at specific offsets within ptr array
    for (unsigned int i = offset; i < num_elements; i += MOD_SZ) {
        ptr[i] = p1;
    }

    // Write p2 at other positions while avoiding division by MOD_SZ multiple times
    for (unsigned int i = 0; i < num_elements; i++) {
        if (i % MOD_SZ != offset) {
            ptr[i] = p2;
        }
    }
}