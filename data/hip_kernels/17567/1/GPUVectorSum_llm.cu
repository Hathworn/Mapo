#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GPUVectorSum(int * a, int * b, int * c, int VECTOR_QNT) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Pre-fetching to utilize shared memory for faster access
    extern __shared__ int sharedA[];

    // Load elements collaboratively
    if (idx < VECTOR_QNT) {
        sharedA[threadIdx.x] = a[idx];
        __syncthreads();  // Ensure all threads have loaded data
        
        // Perform addition with shared memory
        c[idx] = sharedA[threadIdx.x] + b[idx];
    }
}