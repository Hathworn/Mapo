#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function with check for valid thread index
__global__ void helloFromGPU()
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index

    if (idx == 0) // Only one thread prints the message
    {
        printf("Hello, World from GPU!\n");
    }
}