#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convdw_gpu_kernel(float *dw, float *dy, float *x, const int S, const int outSize, const int inSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to optimize access patterns, reducing global memory transactions
    extern __shared__ float sharedMemory[];
    float *sharedX = sharedMemory;
    float *sharedDy = &sharedMemory[S * blockDim.y];

    if (row < inSize && col < outSize) {
        float dwValue = 0.0f; // Accumulate partial results in register

        for (int i = 0; i < S; ++i) {
            if (threadIdx.y < blockDim.y) {
                sharedX[threadIdx.y + i * blockDim.y] = x[row + S * i];
                sharedDy[threadIdx.x + i * blockDim.x] = dy[i * outSize + col];
            }
            __syncthreads(); // Ensure shared memory is populated for all threads

            // Compute with values from shared memory
            dwValue += sharedX[threadIdx.y + i * blockDim.y] * sharedDy[threadIdx.x + i * blockDim.x];
            __syncthreads(); // Ensure completion before next iteration
        }

        dw[row * outSize + col] += dwValue; // Update result in global memory
    }
}