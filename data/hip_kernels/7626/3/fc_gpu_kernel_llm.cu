#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fc_gpu_kernel(float *y, float *x, float *weights, const int weightHeight, const int outSize, const int inSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < inSize && col < outSize) {
        float acc = 0;  // Use an accumulator to avoid repeated memory access
        for (int i = 0; i < weightHeight; ++i) {
            acc += x[row * weightHeight + i] * weights[i * outSize + col];
        }
        y[row * outSize + col] = acc;
    }
}