#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convdx_gpu_kernel(float *dx, float *dy, float *weights, const int S, const int outSize, const int inSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if we are within bounds
    if (row < inSize && col < outSize) {
        float sum = 0.0f;  // Accumulate result in a local variable

        // Loop unrolling for improved performance
        for (int i = 0; i < S; i += 4) {
            if (i + 0 < S) sum += dy[row * S + (i + 0)] * weights[col * S + (i + 0)];
            if (i + 1 < S) sum += dy[row * S + (i + 1)] * weights[col * S + (i + 1)];
            if (i + 2 < S) sum += dy[row * S + (i + 2)] * weights[col * S + (i + 2)];
            if (i + 3 < S) sum += dy[row * S + (i + 3)] * weights[col * S + (i + 3)];
        }

        dx[row * outSize + col] = sum;  // Write result to output
    }
}