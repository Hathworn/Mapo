#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    // Optimize: Use built-in function for fast index computation
    unsigned int sequenceNumber = blockIdx.x * blockDim.x + threadIdx.x; 

    /* Each thread gets a different sequence number */
    hiprand_init(seed, sequenceNumber, 0, &state[sequenceNumber]);
}