#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    extern __shared__ float shmem[];  // Use dynamic shared memory to reduce shared memory size the kernel uses

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float tempSum = 0.0f;  // Use register to store partial sum for each thread
    if (eidx < numElements) {
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            tempSum += a[eidx] * b[eidx];
        }
    }
    shmem[threadIdx.x] = tempSum;  // Store the register result to shared memory
    __syncthreads();

    // Unrolled reduction in shared memory
    if (threadIdx.x < 128) { shmem[threadIdx.x] += shmem[threadIdx.x + 128]; } __syncthreads();
    if (threadIdx.x < 64) { shmem[threadIdx.x] += shmem[threadIdx.x + 64]; } __syncthreads();
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;  // Use volatile pointer to prevent load/store optimizations
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }
    
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];  // Write block's result to target
    }
}