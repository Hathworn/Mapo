#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory for synchronization among threads
    __shared__ float shared_accuracy;
    
    if (threadIdx.x == 0) {
        // Only one thread does the division to avoid race conditions
        shared_accuracy = *accuracy / N;
    }
    __syncthreads(); // Synchronize threads to ensure shared memory update

    if (threadIdx.x == 0) {
        // Only one thread updates the global memory
        *accuracy = shared_accuracy;
    }
}