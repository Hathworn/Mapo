#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
  const int tx = blockIdx.x * blockDim.x + threadIdx.x;  // Use blockDim.x for better flexibility and grid size calculation

  if (tx < numCases) {
    const int label = static_cast<int>(labels[tx]);  // Use static_cast for type safety
    const float maxp = maxProbs[tx];
    const float labelp = __ldg(&probs[label * numCases + tx]);  // Use __ldg for faster memory access

    labelLogProbs[tx] = __logf(labelp);

    if (labelp != maxp) {
      correctProbs[tx] = 0.0f;
    } else {
      int numMax = 0;
      for (int i = 0; i < numOut; i++) {
        numMax += (__ldg(&probs[i * numCases + tx]) == maxp);  // Use __ldg inside the loop for memory access optimization
      }
      correctProbs[tx] = 1.0f / static_cast<float>(numMax);  // Static cast for consistent type
    }
  }
}