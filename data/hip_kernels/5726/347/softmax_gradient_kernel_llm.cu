#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    // Offset pointers for this block
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    
    // Thread index and shared memory
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // First level of reduction: each thread computes partial sum
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Second level of reduction: accumulate results across all threads
    if (idx == 0) {
        float sum = 0.0f;
        for (int i = 0; i < blockDim.x; ++i) {
            sum += reduction_buffer[i];
        }
        reduction_buffer[0] = sum;
    }
    __syncthreads();

    // Compute the gradients using the inner product result
    const float sum = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - sum);
    }
}