#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_image2D1C_ConvolveColumn(float* img, int n_x, int n_y, short k, float *kernel, float* out)
{
    // Find index of current thread
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check within bounds
    if (idx_x < n_x && idx_y < n_y) {
        float sum = 0;
        // Loop unrolling for optimization
        #pragma unroll
        for (short i = -k; i <= k; ++i) {
            short y = idx_y + i;
            if (y < 0) y = 0;
            if (y >= n_y) y = n_y - 1;
            sum += kernel[i + k] * img[y * n_x + idx_x];
        }
        // Store the result
        out[idx_y * n_x + idx_x] = sum;
    }
}