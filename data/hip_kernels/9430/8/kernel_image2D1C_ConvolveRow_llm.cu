#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_image2D1C_ConvolveRow(float* img, int n_x, int n_y, short k, float *kernel, float* out)
{
    // Calculate linear index in x and y dimensions
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Return early if indices are out of bounds
    if (idx_x >= n_x || idx_y >= n_y) return;

    float sum = 0;
    // Cache kernel half-width for re-use in loop condition
    short kernel_offset = k;
    
    for (short i = -kernel_offset; i <= kernel_offset; i++)
    {
        // Optimize boundary checks using max and min functions
        short x = max(0, min(n_x - 1, idx_x + i));
        
        // Accumulate convolution sum
        sum += kernel[i + kernel_offset] * img[idx_y * n_x + x];
    }
    
    // Store result to output array
    out[idx_y * n_x + idx_x] = sum;
}