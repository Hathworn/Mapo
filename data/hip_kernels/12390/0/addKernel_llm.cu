#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//#define array_size 100000000
#define array_size 101

hipError_t addWithCuda(int *total);

__shared__ int temp[array_size];

__global__ void addKernel(int *tid_c, int *tid_total)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < array_size)
    {
        tid_c[tid] = tid;
        temp[threadIdx.x] = tid;

        // Use __syncthreads to ensure all threads have written to shared memory
        __syncthreads();

        // Only thread 0 in each block performs the atomic addition
        if (threadIdx.x == 0)
        {
            int block_sum = 0;
            for (int i = 0; i < blockDim.x; i++)
            {
                // Accumulate current block's contribution
                block_sum += temp[i];
            }
            // Perform atomic addition with the accumulated block sum
            atomicAdd(tid_total, block_sum);
        }
    }
}