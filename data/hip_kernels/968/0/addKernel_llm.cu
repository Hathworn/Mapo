#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global index for the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use conditional to ensure that only valid indices are processed
    if (i < gridDim.x * blockDim.x) {
        c[i] = a[i] + b[i];
    }
}