#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSign(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Unroll the loop to improve performance.
    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = (mat[i] > 0) - (mat[i] < 0); // Optimize copysignf with a faster operation
    }
}