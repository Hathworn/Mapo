#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddRowMult(float* mat, float* vec, float* tgtMat, float mult, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        // Calculate row index once for each loop iteration
        unsigned int rowIndex = i / height;
        tgtMat[i] = mat[i] + mult * vec[rowIndex];
    }
}
```
