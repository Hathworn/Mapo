#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCorrectPreds(float* mat, float* p, float* target, unsigned int len, float cutoff) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use coalesced memory reads and stride step
    for (unsigned int i = idx; i < len; i += stride) {
        float matVal = mat[i];
        float pVal = p[i];
        target[i] = matVal * (pVal >= cutoff) + (1 - matVal) * (pVal < cutoff);
    }
}