#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplyAbs(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    // Loop unrolling optimization
    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = fabsf(mat[i]);
    }
}