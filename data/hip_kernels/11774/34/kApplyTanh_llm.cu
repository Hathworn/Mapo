#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kApplyTanh(float* mat, float* target, unsigned int len) {
    // Use shared memory to optimize memory access
    extern __shared__ float shared_mat[];
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        // Load data into shared memory for coalesced access
        shared_mat[threadIdx.x] = mat[i];
        __syncthreads();

        // Compute Tanh using shared memory data
        float mat_i = shared_mat[threadIdx.x];
        float exp2x = __expf(2 * mat_i);
        target[i] = 1 - 2 / (exp2x + 1);
        
        __syncthreads();
    }
}