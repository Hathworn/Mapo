#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kMultDiagonalScalar(float* mat, float val, float* tgtMat, unsigned int width) {
    // Optimize by calculating stride for better memory access coalescing
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    for (unsigned int i = idx; i < width; i += stride) {
        // Access matrix diagonally and perform multiplication
        const unsigned int elementIndex = width * i + i;
        tgtMat[elementIndex] = mat[elementIndex] * val;
    }
}