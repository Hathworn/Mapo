#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width) { // Boundary check for valid rows and columns
        tgtMat[row * width + col] = mat[row * width + col] + vec[row]; // Direct indexing for row and column
    }
}