#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Compute matrix row and column using thread index directly to avoid unnecessary calculations
    const unsigned int row = (blockIdx.x * blockDim.x + threadIdx.x) / width;
    const unsigned int col = (blockIdx.x * blockDim.x + threadIdx.x) % width;
    // Ensure index is within bounds and avoid bank conflicts by accessing memory in contiguous blocks
    if (row < height) {
        tgtMat[row * width + col] = mat[row * width + col] * vec[row];
    }
}