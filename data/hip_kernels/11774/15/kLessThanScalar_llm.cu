#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLessThanScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by removing unnecessary loop and streamline calculations
    if (idx < len) {
        target[idx] = mat[idx] < val;
    }
}