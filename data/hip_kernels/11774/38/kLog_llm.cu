#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLog(float* mat, float* target, unsigned int len, float tiny) {
    // Calculate global index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we don't go out of bounds
    if (idx < len) {
        target[idx] = __logf(mat[idx] + tiny);
    }
}