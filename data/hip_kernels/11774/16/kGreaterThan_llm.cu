#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kGreaterThan(float* __restrict__ mat1, float* __restrict__ mat2, float* __restrict__ target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < len) {
        // Determine if mat1 element is greater than mat2 element
        target[idx] = mat1[idx] > mat2[idx];
    }
}