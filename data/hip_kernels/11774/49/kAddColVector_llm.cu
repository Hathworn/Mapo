#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate the row and column index for each thread
    const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        unsigned int index = row * width + col; // Compute linear index
        // Perform the addition operation
        tgtMat[index] = mat[index] + vec[row];
    }
}
```
