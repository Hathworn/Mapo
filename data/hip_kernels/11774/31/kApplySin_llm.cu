#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplySin(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Loop unrolling for performance optimization
    for (unsigned int i = idx; i < len; i += stride * 4) {
        target[i] = __sinf(mat[i]);
        if (i + stride < len) target[i + stride] = __sinf(mat[i + stride]);
        if (i + 2 * stride < len) target[i + 2 * stride] = __sinf(mat[i + 2 * stride]);
        if (i + 3 * stride < len) target[i + 3 * stride] = __sinf(mat[i + 3 * stride]);
    }
}