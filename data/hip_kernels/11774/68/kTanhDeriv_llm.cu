#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTanhDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    // Loop with stride for coalesced global memory access and better occupancy
    for (unsigned int i = idx; i < numEls; i += stride) {
        float bi = b[i]; // Cache b[i] to reduce memory reads
        dest[i] = a[i] * (1.0f + bi) * (1.0f - bi) * 0.5f; // Use float literals
    }
}