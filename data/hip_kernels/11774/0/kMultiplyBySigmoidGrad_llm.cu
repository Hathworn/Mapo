#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultiplyBySigmoidGrad(float* __restrict__ act, float* __restrict__ target, const unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    for(unsigned int i = idx; i < len; i += stride) {
        float act_val = act[i];  // Load act value once
        target[i] *= act_val * (1.0f - act_val);  // Reduce repetitive access
    }
}