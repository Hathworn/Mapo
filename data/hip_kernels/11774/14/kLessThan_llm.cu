#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLessThan(float* mat1, float* mat2, float* target, unsigned int len) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize by ensuring a clean boundary check within the loop
    if (idx < len) {
        target[idx] = mat1[idx] < mat2[idx];
    }
}