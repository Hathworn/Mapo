#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate flat index for the matrix element
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop with grid-stride to cover all elements
    for (unsigned int i = idx; i < width * height; i += gridDim.x * blockDim.x) {
        // Efficient indexing to access columns divided by corresponding vector element
        tgtMat[i] = mat[i] / vec[i % height];
    }
}