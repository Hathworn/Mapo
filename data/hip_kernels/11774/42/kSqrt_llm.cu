#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSqrt(float* mat, float* target, unsigned int len) {
    // Calculate the index for this thread
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop over each element this thread should process.
    // Use stride to ensure all elements are covered by all threads.
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        target[i] = sqrtf(mat[i]);  // Use sqrtf for single precision float
    }
}