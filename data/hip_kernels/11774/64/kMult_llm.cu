#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kMult(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Use stride loop for better memory access pattern
    for (unsigned int i = idx; i < numEls; i += blockDim.x * gridDim.x) {
        dest[i] = a[i] * b[i];
    }
}