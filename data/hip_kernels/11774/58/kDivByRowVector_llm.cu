#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    const unsigned int rowStart = (idx / width) * width; // Pre-calculate row start index

    // Optimize loop by reducing number of arithmetic operations
    for (unsigned int i = idx; i < width * height; i += numThreads) {
        unsigned int rowIndex = i / width;
        tgtMat[i] = mat[i] / vec[rowIndex];
    }
}