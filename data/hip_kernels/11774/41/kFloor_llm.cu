#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kFloor(float* __restrict__ mat, float* __restrict__ target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Unrolling loop to improve performance
    for (unsigned int i = idx; i < len; i += numThreads * 4) {
        if (i < len) target[i] = floorf(mat[i]);
        if (i + numThreads < len) target[i + numThreads] = floorf(mat[i + numThreads]);
        if (i + 2 * numThreads < len) target[i + 2 * numThreads] = floorf(mat[i + 2 * numThreads]);
        if (i + 3 * numThreads < len) target[i + 3 * numThreads] = floorf(mat[i + 3 * numThreads]);
    }
}