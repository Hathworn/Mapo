#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kReciprocal(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Iterate through elements with stride for better memory access
    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = 1.0f / mat[i]; // Use 1.0f for better precision with floats
    }
}