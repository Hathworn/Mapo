#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAssignScalar(float* dest, float alpha, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Utilize memory coalescing by using consecutive threads to write data continuously
    for (unsigned int i = idx; i < len; i += stride) {
        dest[i] = alpha;
    }
}