#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLowerBoundScalar(float* mat, float val, float* target, unsigned int len) {
    // Use a separate local variable to store calculated index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    
    // Loop incremented based on numThreads, reducing divergence
    for (unsigned int i = idx; i < len; i += numThreads) {
        float m = mat[i];
        target[i] = m < val ? val : m;
    }
}