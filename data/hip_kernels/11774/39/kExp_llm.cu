#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kExp(float* __restrict__ mat, float* __restrict__ target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Optimize memory access pattern and reduce register pressure
    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = __expf(mat[i]);
    }
}