#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddDiagonalScalar(float* mat, float val, float* tgtMat, unsigned int width) {
    // Use a single loop for the exact diagonal required for current thread
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure only threads within bounds update the diagonal
    if (i < width) {
        tgtMat[width * i + i] = mat[width * i + i] + val;
    }
}