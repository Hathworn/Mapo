#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBlockify(float* source, float* target, int numdims, int blocksize) {
    const unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Use global thread index
    const unsigned int numThreads = gridDim.x * blockDim.x;          // Compute total threads in the grid

    if (idx < numdims) {                                              // Check if thread index within bounds
        const int block = idx / blocksize;
        target[idx] = source[block * blocksize];
    }
}