#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kUpperBound(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure only active threads do computations
    if (idx < len) {
        target[idx] = mat1[idx] > mat2[idx] ? mat2[idx] : mat1[idx];
    }
}