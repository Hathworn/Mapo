#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Utilize shared memory to reduce global memory access for vec
    extern __shared__ float sharedVec[];
    const unsigned int tx = threadIdx.x;

    // Load vector data into shared memory
    if (tx < height) {
        sharedVec[tx] = vec[tx];
    }
    __syncthreads();

    const unsigned int idx = blockIdx.x * blockDim.x + tx;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        tgtMat[i] = mat[i] * sharedVec[i % height];
    }
}