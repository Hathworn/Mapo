#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddScalar(float* a, float alpha, float* dest, unsigned int numEls) {
    // Use a single thread stride loop for improved warp efficiency
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numEls; i += blockDim.x * gridDim.x) {
        dest[i] = a[i] + alpha;
    }
}