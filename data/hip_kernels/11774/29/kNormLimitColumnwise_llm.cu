#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use a defined constant for the block size
#define BLOCK_SIZE 32

__global__ void kNormLimitColumnwise(float* mat, float* target, float norm, unsigned int width, unsigned int height) {
    // Use the shared memory only up to needed index
    __shared__ float sum_vals[BLOCK_SIZE + 1];

    float cur_sum = 0.0f;

    // Use block size constant
    for (unsigned int i = threadIdx.x; i < height; i += BLOCK_SIZE) {
        float val = mat[blockIdx.x * height + i];
        cur_sum += val * val;
    }
    sum_vals[threadIdx.x] = cur_sum;

    __syncthreads();

    // Unroll the loop for performance improvement
    if (threadIdx.x == 0) {
        cur_sum = 0.0f;
        for (unsigned int i = 0; i < BLOCK_SIZE; i += 4) {
            cur_sum += sum_vals[i];
            cur_sum += sum_vals[i + 1];
            cur_sum += sum_vals[i + 2];
            cur_sum += sum_vals[i + 3];
        }
        // Minimize synchronization points
        sum_vals[BLOCK_SIZE] = norm > cur_sum ? 1.0f : norm / sqrt(cur_sum);
    }

    __syncthreads();

    float scale = sum_vals[BLOCK_SIZE];

    for (unsigned int i = threadIdx.x; i < height; i += BLOCK_SIZE) {
        target[blockIdx.x * height + i] = mat[blockIdx.x * height + i] * scale;
    }
}