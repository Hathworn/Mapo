#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kGenerateTranslationsBigVarOff(float* source, float* target, float* off_x_arr, float* off_y_arr, int source_w, int target_w, int num_channels) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; // Flatten the thread index
    const unsigned int stride = blockDim.x * gridDim.x; // Calculate total stride

    int pad = (source_w - target_w) / 2;
    int target_tile_size = target_w * target_w;
    int source_tile_size = source_w * source_w;

    int off_x = off_x_arr[blockIdx.x];
    int off_y = off_y_arr[blockIdx.x];

    while (idx < target_tile_size) { // Use 'while' for better loop handling
        int target_x = idx / target_w;
        int target_y = idx % target_w; // Simplify calculation with modulo operator

        int target_index = target_off + target_x * target_w + target_y;
        int source_index = source_off + target_x * source_w + target_y;

        #pragma unroll // Suggest the compiler to unroll the loop
        for (unsigned int ch = 0; ch < num_channels; ch++) {
            target[num_channels * target_index + ch] =
                source[num_channels * source_index + ch];
        }

        idx += stride; // Increment by stride
    }
}