#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd(float* a, float* b, float* dest, unsigned int numEls) {
    // Compute global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll loop to increase throughput
    for (unsigned int i = idx; i < numEls; i += blockDim.x * gridDim.x) {
        dest[i] = a[i] + b[i];
    }
}