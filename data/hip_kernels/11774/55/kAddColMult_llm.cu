#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddColMult(float* mat, float* vec, float* tgtMat, float mult, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    const unsigned int stride = numThreads;

    // Use a while loop for fewer instructions and better performance
    unsigned int i = idx;
    while (i < width * height) {
        tgtMat[i] = mat[i] + mult * vec[i % height];
        i += stride;
    }
}