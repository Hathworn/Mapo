#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplyLog1PlusExp(float* mat, float* target, unsigned int len) {
    extern __shared__ float sharedMem[]; // Use shared memory for efficient access
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    
    for (unsigned int i = idx; i < len; i += numThreads) {
        sharedMem[threadIdx.x] = mat[i]; // Cache data in shared memory
        float mat_i = sharedMem[threadIdx.x];
        // Simplify branching and use intrinsic logf and expf
        float exp_val = __expf(mat_i > 0 ? -mat_i : mat_i);
        target[i] = __logf(1 + exp_val) + (mat_i > 0 ? mat_i : 0);
    }
}