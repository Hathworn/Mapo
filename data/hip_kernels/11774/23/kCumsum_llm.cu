#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCumsum(float *mat, float *target, float *temp, unsigned int height) {
    const int thid = threadIdx.x;
    const int super_offset = blockIdx.x * height;

    // Adjust pointers for block offset
    target += super_offset;
    mat += super_offset;
    temp += super_offset;

    if (2 * thid < height) {
        int offset = 1;
        
        // Load input into shared memory
        temp[2 * thid] = mat[2 * thid];
        temp[2 * thid + 1] = mat[2 * thid + 1];

        // Build sum in place up the tree
        for (int d = height >> 1; d > 0; d >>= 1) {
            __syncthreads();
            if (thid < d) {
                int ai = offset * (2 * thid + 1) - 1;
                int bi = offset * (2 * thid + 2) - 1;
                temp[bi] += temp[ai];
            }
            offset *= 2;
        }

        // Clear the last element
        if (thid == 0) { temp[height - 1] = 0; }

        // Traverse down tree & build scan
        for (int d = 1; d < height; d *= 2) {
            offset >>= 1;
            __syncthreads();
            if (thid < d) {
                int ai = offset * (2 * thid + 1) - 1;
                int bi = offset * (2 * thid + 2) - 1;
                float t = temp[ai];
                temp[ai] = temp[bi];
                temp[bi] += t;
            }
        }
        __syncthreads();

        // Write results to device memory
        target[2 * thid] = temp[2 * thid];
        target[2 * thid + 1] = temp[2 * thid + 1];
    }
}