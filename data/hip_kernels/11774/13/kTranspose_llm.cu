#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTranspose(float *odata, float *idata, int width, int height) {
    __shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE + 1];

    // Calculate indices
    unsigned int xIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
    unsigned int yIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;

    // Read matrix tile into shared memory
    if ((xIndex < width) && (yIndex < height)) {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }

    __syncthreads();

    // Calculate transposed indices
    xIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
    yIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;

    // Write transposed matrix tile to global memory
    if ((xIndex < height) && (yIndex < width)) {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}