#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivide(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEls) { // Optimize: check bounds before accessing memory
        dest[idx] = a[idx] / b[idx];
    }
}