#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLowerBound(float* mat1, float* mat2, float* target, unsigned int len) {
    // Calculate global thread ID
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop to efficiently utilize GPU resources
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        // Use max function for clarity and potential performance gain
        target[i] = fmaxf(mat1[i], mat2[i]);
    }
}