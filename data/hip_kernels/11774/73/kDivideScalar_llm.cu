#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivideScalar(float* mat, float alpha, float* dest, unsigned int len) {
    // Use a single ID calculation to reduce redundancy
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride looping to improve memory coalescing
    unsigned int stride = gridDim.x * blockDim.x;
    while (i < len) {
        dest[i] = mat[i] / alpha;
        i += stride;
    }
}