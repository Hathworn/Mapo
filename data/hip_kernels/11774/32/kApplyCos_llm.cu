#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplyCos(float* __restrict__ mat, float* __restrict__ target, unsigned int len) {
    // Calculate the global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single-step loop with stride of total number of threads
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        target[i] = __cosf(mat[i]);
    }
}