#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSetSelectedRows(float* target, float* source, float* indices, int nRowIs, int nCols, int nTargetRows){
    __shared__ int targetRowIndices[32];
    const int startSourceRowI = blockIdx.x * 32;
    const int tid = threadIdx.x;
    const int localNRowIs = min(32, nRowIs-startSourceRowI);

    // Load row indices in parallel
    if (tid < localNRowIs) {
        int index = int(indices[startSourceRowI + tid]);
        if (index < 0) index += nTargetRows; 
        targetRowIndices[tid] = (index < 0 || index >= nTargetRows) ? -1 : index;
    }
    __syncthreads();

    // Optimized row copy using thread block parallelism
    for (int i = 0; i < localNRowIs; i++) {
        const int targetRowI = targetRowIndices[i];
        if (targetRowI != -1) {  // Skip invalid rows
            for (int colI = tid; colI < nCols; colI += 32) {
                target[targetRowI * nCols + colI] = source[(startSourceRowI + i) * nCols + colI];
            }
        } else if (tid < nCols) {  // Set invalid target indices to NaN for the first warp
            target[tid + i * nCols] = (1.0 / 0.0 - 1.0 / 0.0);
        }
    }
}