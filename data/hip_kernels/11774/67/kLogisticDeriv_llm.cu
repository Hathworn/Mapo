#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogisticDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Use shared memory to reduce global memory access
    __shared__ float shared_a[256]; // Adjust the size based on blockDim.x
    __shared__ float shared_b[256];

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        shared_a[threadIdx.x] = a[i];
        shared_b[threadIdx.x] = b[i];

        __syncthreads(); // Ensure all threads have loaded the data

        // Perform computation using shared memory
        dest[i] = shared_a[threadIdx.x] * shared_b[threadIdx.x] * (1.0 - shared_b[threadIdx.x]);

        __syncthreads(); // Prevent race conditions on shared memory
    }
}