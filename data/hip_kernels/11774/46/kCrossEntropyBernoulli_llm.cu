#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCrossEntropyBernoulli(float* mat, float* p, float* target, unsigned int len, float tiny) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Loop unrolling for better performance
    for (unsigned int i = idx; i < len; i += stride * 4) {
        if (i < len) {
            target[i] = -mat[i] * __logf(p[i] + tiny) - (1 - mat[i]) * __logf(1 - p[i] + tiny);
        }
        if (i + stride < len) {
            target[i + stride] = -mat[i + stride] * __logf(p[i + stride] + tiny) - (1 - mat[i + stride]) * __logf(1 - p[i + stride] + tiny);
        }
        if (i + 2 * stride < len) {
            target[i + 2 * stride] = -mat[i + 2 * stride] * __logf(p[i + 2 * stride] + tiny) - (1 - mat[i + 2 * stride]) * __logf(1 - p[i + 2 * stride] + tiny);
        }
        if (i + 3 * stride < len) {
            target[i + 3 * stride] = -mat[i + 3 * stride] * __logf(p[i + 3 * stride] + tiny) - (1 - mat[i + 3 * stride]) * __logf(1 - p[i + 3 * stride] + tiny);
        }
    }
}