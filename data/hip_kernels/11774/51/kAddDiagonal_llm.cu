#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddDiagonal(float* mat, float* vec, float* tgtMat, unsigned int width) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Loop with stride to utilize all threads efficiently
    for (unsigned int i = idx; i < width; i += stride) {
        tgtMat[width * i + i] = mat[width * i + i] + vec[i];
    }
}