#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSwapRows(float* source, float* target, float* indices1, float* indices2, int nRowIs, int nCols, int nRows){
    __shared__ int sourceRowIndices[32], targetRowIndices[32];
    const int startRowI = blockIdx.x * 32;
    const int tid = threadIdx.x;
    const int localNRowIs = min(32, nRowIs - startRowI);

    // Load 32 row indices cooperatively
    if (tid < localNRowIs) {
        int srcIdx = int(indices1[startRowI + tid]);
        int tgtIdx = int(indices2[startRowI + tid]);

        if (srcIdx < 0) srcIdx += nRows;
        if (tgtIdx < 0) tgtIdx += nRows;

        sourceRowIndices[tid] = (srcIdx < 0 || srcIdx >= nRows) ? -1 : srcIdx;
        targetRowIndices[tid] = (tgtIdx < 0 || tgtIdx >= nRows) ? -1 : tgtIdx;
    }
    __syncthreads();

    // Copy rows and swap them
    for (int i = 0; i < localNRowIs; i++) {
        const int sourceRowI = sourceRowIndices[i], targetRowI = targetRowIndices[i];
        for (int colI = tid; colI < nCols; colI += 32) {
            const float temp1 = (sourceRowI == -1) ? 0.0f : source[sourceRowI * nCols + colI];
            const float temp2 = (targetRowI == -1) ? 0.0f : target[targetRowI * nCols + colI];
            
            if (sourceRowI != -1) source[sourceRowI * nCols + colI] = temp2;
            if (targetRowI != -1) target[targetRowI * nCols + colI] = temp1;
        }
    }
}