#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kRectifiedLinearDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate the global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll the loop to handle multiple elements per thread for better performance
    const unsigned int stride = blockDim.x * gridDim.x;
    unsigned int i = idx;
    
    while (i < numEls) {
        dest[i] = a[i] * (b[i] > 0 ? 1 : 0);
        i += stride; // Move to the next element handled by this thread
    }
}