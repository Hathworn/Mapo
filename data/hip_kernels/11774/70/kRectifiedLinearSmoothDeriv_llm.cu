#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kRectifiedLinearSmoothDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by using a single loop counter
    for (unsigned int i = threadId; i < numEls; i += blockDim.x * gridDim.x) {
        dest[i] = a[i] * (1 - __expf(-b[i])); // Improved readability
    }
}