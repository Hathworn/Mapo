#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSinDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEls) { // Optimize by checking boundary condition once
        dest[idx] = a[idx] * __cosf(b[idx]);
    }
}