#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kApplyLog1PlusExpExact(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    float mat_i;

    // Loop unrolling for better performance
    for (unsigned int i = idx; i < len; i += numThreads * 4) {
        if (i < len) {
            mat_i = mat[i];
            target[i] = (mat_i > 0) ? (log(1 + exp(-mat_i)) + mat_i) : log(1 + exp(mat_i));
        }
        if (i + numThreads < len) {
            mat_i = mat[i + numThreads];
            target[i + numThreads] = (mat_i > 0) ? (log(1 + exp(-mat_i)) + mat_i) : log(1 + exp(mat_i));
        }
        if (i + 2*numThreads < len) {
            mat_i = mat[i + 2*numThreads];
            target[i + 2*numThreads] = (mat_i > 0) ? (log(1 + exp(-mat_i)) + mat_i) : log(1 + exp(mat_i));
        }
        if (i + 3*numThreads < len) {
            mat_i = mat[i + 3*numThreads];
            target[i + 3*numThreads] = (mat_i > 0) ? (log(1 + exp(-mat_i)) + mat_i) : log(1 + exp(mat_i));
        }
    }
}