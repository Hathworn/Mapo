#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCrossEntropy(float* mat, float* p, float* target, unsigned int len, float tiny) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += stride) {
        // Optimize by using native math function and avoiding redundant computation
        target[i] = -mat[i] * __logf(p[i] + tiny);
    }
}