#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kPowMatrix(float* mat, float* pow, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop to effectively utilize all threads
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        target[i] = powf(mat[i], pow[i]);
    }
}