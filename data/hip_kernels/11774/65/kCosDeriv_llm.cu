#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCosDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    // Cache thread-specific index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread index does not exceed the number of elements
    if (idx < numEls) {
        dest[idx] = -a[idx] * __sinf(b[idx]); // Compute directly without loop
    }
}