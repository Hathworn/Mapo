#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCeil(float* mat, float* target, unsigned int len) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds for coalesced memory access
    if (idx < len) {
        target[idx] = ceil(mat[idx]);
    }
}