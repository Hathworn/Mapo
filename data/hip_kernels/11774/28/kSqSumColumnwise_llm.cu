#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSqSumColumnwise(float* mat, float* target, unsigned int width, unsigned int height) {
    __shared__ float sum_vals[32];
    float cur_sum = 0;
    
    // Compute start index for the row
    unsigned int start_index = blockIdx.x * height + threadIdx.x;
    
    // Improved loop increment to ensure consistent accumulation across threads
    for (unsigned int i = start_index; i < blockIdx.x * height + height; i += blockDim.x) {
        cur_sum += mat[i] * mat[i];
    }

    // Store partial results in shared memory
    sum_vals[threadIdx.x] = cur_sum;
    
    __syncthreads();
    
    // Reduce within the block threads
    if (threadIdx.x < 16) {
        sum_vals[threadIdx.x] += sum_vals[threadIdx.x + 16];
    }
    __syncthreads();
    if (threadIdx.x < 8) {
        sum_vals[threadIdx.x] += sum_vals[threadIdx.x + 8];
    }
    __syncthreads();
    if (threadIdx.x < 4) {
        sum_vals[threadIdx.x] += sum_vals[threadIdx.x + 4];
    }
    __syncthreads();
    if (threadIdx.x < 2) {
        sum_vals[threadIdx.x] += sum_vals[threadIdx.x + 2];
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        sum_vals[0] += sum_vals[1];
        target[blockIdx.x] = sum_vals[0];
    }
}