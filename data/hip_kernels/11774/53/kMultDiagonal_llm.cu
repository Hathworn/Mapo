#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultDiagonal(float* mat, float* vec, float* tgtMat, unsigned int width) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < width) { // Ensure only necessary threads are used
        tgtMat[width * idx + idx] = mat[width * idx + idx] * vec[idx];
    }
}