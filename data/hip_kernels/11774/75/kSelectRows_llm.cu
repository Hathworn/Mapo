#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSelectRows(float* __restrict__ source, float* __restrict__ target, float* __restrict__ indices, int nRowIs, int nCols, int nSourceRows) {
    __shared__ int sourceRowIndices[32];
    const int startTargetRowI = blockIdx.x * 32;
    const int tid = threadIdx.x;
    const int localNRowIs = min(32, nRowIs - startTargetRowI);
    
    // Load row indices into shared memory
    if (tid < localNRowIs) {
        int index = int(indices[startTargetRowI + tid]);
        if (index < 0) index += nSourceRows;
        sourceRowIndices[tid] = (index < 0 || index >= nSourceRows) ? -1 : index;
    }
    __syncthreads();
    
    // Copy rows, optimized to avoid unnecessary calculations
    if (tid < nCols) {
        for (int i = 0; i < localNRowIs; i++) {
            const int targetRowI = startTargetRowI + i;
            const int sourceRowI = sourceRowIndices[i];
            if (sourceRowI != -1) {
                target[targetRowI * nCols + tid] = source[sourceRowI * nCols + tid];
            } else {
                target[targetRowI * nCols + tid] = NAN; // Use NaN for invalid source row
            }
        }
    }
}