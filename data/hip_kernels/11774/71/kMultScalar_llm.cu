#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kMultScalar(float* mat, float alpha, float* dest, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Unrolling loop for improved performance
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        dest[i] = alpha * mat[i];
    }
}