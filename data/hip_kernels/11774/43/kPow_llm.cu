#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kPow(float* mat, float pow, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x; // Use stride for clarity

    // Unroll loop for performance improvement
    for (unsigned int i = idx; i < len; i += stride * 4) {
        if (i < len) target[i] = powf(mat[i], pow);
        if (i + stride < len) target[i + stride] = powf(mat[i + stride], pow);
        if (i + 2 * stride < len) target[i + 2 * stride] = powf(mat[i + 2 * stride], pow);
        if (i + 3 * stride < len) target[i + 3 * stride] = powf(mat[i + 3 * stride], pow);
    }
}