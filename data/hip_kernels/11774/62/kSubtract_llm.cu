#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSubtract(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unrolling the loop for better performance
    if (idx < numEls) {
        dest[idx] = a[idx] - b[idx];
    }
}