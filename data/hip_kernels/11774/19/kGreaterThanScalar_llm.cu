#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kGreaterThanScalar(float* mat, float val, float* target, unsigned int len) {
    // Calculate global index for each thread
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure threads operate within bounds of the array
    if (idx < len) {
        target[idx] = mat[idx] > val;
    }
}