#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void padding(int *op, int *ip, int N, int C, int H, int W, int Py, int Px) {
    // Calculate unique thread index across the grid
    unsigned int input_id = blockIdx.x * (gridDim.y * gridDim.z) * blockDim.x + blockIdx.y * (gridDim.z * blockDim.x) + blockIdx.z * blockDim.x + threadIdx.x;

    // Early return if out of bounds
    if (input_id >= N * C * H * W) return;

    // Calculate indices
    int i = input_id / (C * H * W);
    input_id %= (C * H * W);
    int j = input_id / (H * W);
    input_id %= (H * W);
    int k = input_id / W;
    int l = input_id % W;

    // Perform the padding operation
    op[i * C * (H + 2 * Py) * (W + 2 * Px) + j * (H + 2 * Py) * (W + 2 * Px) + (k + Py) * (W + 2 * Px) + (l + Px)] = ip[i * C * H * W + j * H * W + k * W + l];
}