#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lineark(int *ip, int *weight, int *op, int N, int M, int L) {
    unsigned int input_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid computation when input_id is out of bounds
    if (input_id >= N * M * L) return;

    int i = input_id / (M * L);
    int j = (input_id / L) % M;
    int k = input_id % L;

    int temp = ip[i * L + k] * weight[j * L + k];

    // More friendly atomicAdd usage for better performance
    atomicAdd(&op[i * M + j], temp);
}