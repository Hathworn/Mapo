#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv(fmap *input,int *ip,int *weights,int R,int S,fmap *output, int Sx, int Sy,int *op,int Px,int Py) {
    // Use block and thread index for optimized calculation
    unsigned int input_id = (blockIdx.x * gridDim.y + blockIdx.y + blockIdx.z * gridDim.x * gridDim.y) * blockDim.x + threadIdx.x;
    int C = input->dim2;
    int H = input->dim3;
    int W = input->dim4;
    int M = output->dim2;
    int E = output->dim3;
    int F = output->dim4;
    
    H += 2 * Py;
    W += 2 * Px;
    
    // Precompute terms for efficient memory access
    int strideC = F * E * M * R * S;
    int strideM = F * E * R * S;
    int strideE = F * R * S;
    int strideF = R * S;
    int strideR = S;
    
    int i = input_id / strideC;
    input_id %= strideC;
    int j = input_id / strideM;
    input_id %= strideM;
    int k = input_id / strideE;
    input_id %= strideE;
    int l = input_id / strideF;
    input_id %= strideF;
    int m = input_id / (R * S);
    input_id %= (R * S);
    int n = input_id / S;
    int o = input_id % S;

    // Perform computation and atomic addition
    int temp = (*(ip + i * C * H * W + m * H * W + (k * Sy + n) * W + (l * Sx + o))) * (*(weights + j * C * R * S + m * R * S + n * S + o));
    atomicAdd((op + i * M * E * F + j * E * F + k * F + l), temp);
}