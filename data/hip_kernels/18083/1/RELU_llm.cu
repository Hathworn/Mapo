#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RELU(int *ip, int N, int C, int H, int W){
    unsigned int input_id = blockDim.x * blockIdx.x + threadIdx.x;
    if (input_id < N * C * H * W) {  // Ensure within bounds
        int offset = input_id;
        int i = offset / (C * H * W);
        offset %= (C * H * W);
        int j = offset / (H * W);
        offset %= (H * W);
        int k = offset / W;
        int l = offset % W;

        int* addr = ip + i * C * H * W + j * H * W + k * W + l;
        int temp = *addr;
        if (temp < 0)
            *addr = 0;
    }
}