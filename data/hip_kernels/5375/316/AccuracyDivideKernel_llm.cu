#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Improved: Calculate the index of the current thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Improved: Ensure only the first thread performs the division
    if (idx == 0) {
        *accuracy /= N; // Improved: Perform division only once
    }
}