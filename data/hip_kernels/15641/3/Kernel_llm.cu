#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Kernel(int* a, int* b, int* c, int n) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory efficiently with a fixed size
    __shared__ int shared_mem[blockDim.x];
    int reg;

    if (i >= n) return;

    // Load data into registers directly
    reg = a[i] + b[i];
    
    // Store results in shared memory only limited to active threads
    shared_mem[threadIdx.x] = reg;
    __syncthreads();

    // Write back results to global memory
    c[i] = shared_mem[threadIdx.x];
}