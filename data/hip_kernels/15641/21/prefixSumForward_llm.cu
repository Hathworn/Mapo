#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prefixSumForward(float* arr, int step, int n) { // Added n as a parameter
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Combined bx, tx, BX into a single line
    int ii = i + 1; // Direct computation of ii

    // Early exit to reduce unnecessary computations
    if (ii > n || ii <= n / float(step)) return;

    // Improved memory access pattern
    int idx1 = ii * step - 1;
    int idx2 = idx1 - step / 2;
    arr[idx1] += arr[idx2];

    // Avoid boundary condition overhead
    if (step == n && idx1 == n - 1) {
        arr[ii * step] = arr[idx1];
        arr[idx1] = 0.0f; // Ensuring floating point consistency
    }
}