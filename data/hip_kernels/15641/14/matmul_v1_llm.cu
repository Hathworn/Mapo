#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matmul_v1(float* a,float* b,float* c, int n){
// C(nxn) = A(nxn) * B(nxn);

__shared__ float A[TILE_SIZE][TILE_SIZE+1];
__shared__ float B[TILE_SIZE][TILE_SIZE+1];

int bx = blockIdx.x;
int by = blockIdx.y;
int tx = threadIdx.x;
int ty = threadIdx.y;

int i = bx*TILE_SIZE+tx;
int j = by*TILE_SIZE+ty;

// Reset shared memory values
A[ty][tx] = 0;
B[ty][tx] = 0;

if(i >= n || j >= n) return;

float c_ij = 0;
for(int m = 0; m < (n + TILE_SIZE - 1) / TILE_SIZE; m++){

    // Load tile into shared memory if within bounds
    if (tx + m*TILE_SIZE < n && j < n)
        A[ty][tx] = a[j*n + m*TILE_SIZE + tx];
    if (ty + m*TILE_SIZE < n && i < n)
        B[ty][tx] = b[(m*TILE_SIZE + ty)*n + i];

    __syncthreads();

    // Compute on loaded tiles
    for(int k = 0; k < TILE_SIZE; k++)
        c_ij += A[ty][k] * B[k][tx];
    __syncthreads();
}

// Ensure within result bounds before writing
if (i < n && j < n)
    c[n*j+i] = c_ij;

}