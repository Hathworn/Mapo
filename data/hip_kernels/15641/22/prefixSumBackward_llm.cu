#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prefixSumBackward(float* arr, int step, int n) { // Add 'n' as a parameter for boundary check
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Combine index calculation into a single line
    int ii = i + 1;

    if (i >= n || ii * step > n) return; // Fix the boundary condition

    int idx1 = ii * step - 1; // Store index calculations in variables for better readability
    int idx2 = ii * step - step / 2 - 1;

    float temp = arr[idx1];
    arr[idx1] += arr[idx2];
    arr[idx2] = temp;
}