#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blur(int* B, int* G, int* R, int* RB, int* RG, int* RR, int* K, int rows, int cols, int krows, int kcols) {

    // Use shared memory for kernel
    __shared__ int sK[11*11]; // Assuming maximum kernel size of 11x11. Adjust if necessary.
    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    if (tid < krows * kcols) {
        sK[tid] = K[tid];
    }

    __syncthreads();

    int index = blockIdx.x * 1024 + threadIdx.x;

    if (index >= rows * cols) return;

    int pixel_row = index / cols;
    int pixel_col = index - pixel_row * cols;

    int k_center_row = (krows - 1) / 2;
    int k_center_col = (kcols - 1) / 2;

    int k_sum = 0;
    int rb_accum = 0;
    int rg_accum = 0;
    int rr_accum = 0;

    for (int i = 0; i < krows; i++) {
        for (int j = 0; j < kcols; j++) {
            int kr = i - k_center_row;
            int kc = j - k_center_col;

            int pr = pixel_row + kr;
            int pc = pixel_col + kc;

            if (pr >= 0 && pr < rows && pc >= 0 && pc < cols) {
                int idx = pr * cols + pc;
                int k_value = sK[i * kcols + j];

                k_sum += k_value;
                rb_accum += B[idx] * k_value;
                rg_accum += G[idx] * k_value;
                rr_accum += R[idx] * k_value;
            }
        }
    }

    if (k_sum != 0) {
        RB[index] = rb_accum / k_sum;
        RG[index] = rg_accum / k_sum;
        RR[index] = rr_accum / k_sum;
    }
}