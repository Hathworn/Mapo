#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void strided_access(float* a, int s, int max) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int index = idx * s; // Calculate index only once
    if (index < max) { // Change conditional logic for better readability
        a[index] += 1; // Use compound assignment operator
    }
}