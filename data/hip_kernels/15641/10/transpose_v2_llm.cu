#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_v2(float* a, float* b, int n) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Calculate global indices
    int i = bx * BX + tx;
    int j = by * BY + ty;

    __shared__ float tile[BY][BX];

    // Check against bounds and load tile
    if (i < n && j < n) {
        tile[ty][tx] = a[j * n + i];
    }

    __syncthreads();

    // Transpose write
    i = by * BY + tx;
    j = bx * BX + ty;

    // Check against bounds before writing
    if (i < n && j < n) {
        b[j * n + i] = tile[tx][ty];
    }
}