#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_v0(float* in, float* out, int n) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BX = blockDim.x; // same as THREAD_MAX
    int i = bx * BX + tx;

    __shared__ float S[THEAD_MAX];

    // Allocate input to shared memory or 0 if outside bounds
    S[tx] = i < n ? in[i] : 0;
    __syncthreads();

    // Perform tree-based reduction
    for (int s = BX / 2; s > 0; s >>= 1) {
        if (tx < s) {
            S[tx] += S[tx + s];
        }
        __syncthreads();
    }

    // First thread in block writes result to output
    if (tx == 0)
        out[bx] = S[0];
}