#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void warp_reduce(float* S, int tx) {
    // Efficient warp-level reduction with loop unrolling
    if (tx < 32) {
        S[tx] += S[tx + 32];
        S[tx] += S[tx + 16];
        S[tx] += S[tx + 8];
        S[tx] += S[tx + 4];
        S[tx] += S[tx + 2];
        S[tx] += S[tx + 1];
    }
}

__global__ void reduce_v5(float* in, float* out, int n) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int i = bx * (BX * 2) + tx;

    __shared__ float S[BX]; 

    // Coalesced reads and reduce per block
    S[tx] = in[i] + in[i + BX];
    __syncthreads();

    // Use dynamic parallelism based on block size
    if (BX >= 1024) {
        if (tx < 512) S[tx] += S[tx + 512];
        __syncthreads();
    }

    if (BX >= 512) {
        if (tx < 256) S[tx] += S[tx + 256];
        __syncthreads();
    }

    if (BX >= 256) {
        if (tx < 128) S[tx] += S[tx + 128];
        __syncthreads();
    }

    if (BX >= 128) {
        if (tx < 64) S[tx] += S[tx + 64];
        __syncthreads();
    }

    if (tx < 32) {
        // Perform warp-level reduction
        warp_reduce(S, tx);
    }

    // Write block result to global memory
    if (tx == 0) out[bx] = S[0];
}