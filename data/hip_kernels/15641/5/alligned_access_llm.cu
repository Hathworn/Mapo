#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void aligned_access(float* a, int max) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use cooperative threads to fetch a whole warp at a time
    for (int i = idx; i < max; i += blockDim.x * gridDim.x) {
        a[i] = a[i] + 1.0f; // Improved memory coalescing
    }
}