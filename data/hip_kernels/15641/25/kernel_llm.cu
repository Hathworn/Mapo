#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(int* arr, int offset_min, int n) {
    // Use block and thread indices to calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the index is within valid range
    if (i < 0 || i >= n) return;

    // Perform operation
    arr[i + offset_min] += 1;
}