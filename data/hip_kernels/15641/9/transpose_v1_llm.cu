#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_v1(float* a, float* b, int n) {
    // Use shared memory for faster access within a block
    __shared__ float tile[32][32+1]; // +1 to avoid bank conflicts

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int indexIn = x + y * n;

    if (x < n && y < n) {
        // Load data from global to shared memory
        tile[threadIdx.y][threadIdx.x] = a[indexIn];
    }
    __syncthreads();

    // Transpose coordinates within shared memory
    int transposedX = blockIdx.y * blockDim.y + threadIdx.x;
    int transposedY = blockIdx.x * blockDim.x + threadIdx.y;
    int indexOut = transposedX + transposedY * n;

    if (transposedX < n && transposedY < n) {
        // Write transposed data back to global memory
        b[indexOut] = tile[threadIdx.x][threadIdx.y];
    }
}