#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void foo() {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;  // Calculate global thread ID
    __shared__ float sharedPow;                       // Use shared memory
    if (tid == 0) {                                   // Only the first thread computes the shared pow
        sharedPow = pow(2, 32);
    }
    __syncthreads();                                  // Ensure the shared value is loaded

    for (int i = 0; i < 1000; i++) {
        float result = sharedPow;                     // Use precomputed result from shared memory
    }
}