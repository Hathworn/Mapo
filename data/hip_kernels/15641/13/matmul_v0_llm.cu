#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to perform matrix multiplication
__global__ void matmul_v0(float* a, float* b, float* c, int n) {
    // Calculate row and column index for this thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread operates within matrix bounds
    if (i >= n || j >= n) return;

    // Initialize the local register for result accumulation
    float c_ij = 0;

    // Loop to accumulate the dot product
    // Unroll the loop manually to optimize
    #pragma unroll
    for (int k = 0; k < n; k++) {
        c_ij += a[n * j + k] * b[n * k + i];
    }

    // Store computed value in the result matrix
    c[n * j + i] = c_ij;
}