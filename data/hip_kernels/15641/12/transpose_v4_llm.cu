#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_v4(float* a, float* b, int n) {
    // Calculate transposed block index
    int blockIdx_x = blockIdx.y;
    int blockIdx_y = (blockIdx.x + blockIdx.y) % gridDim.x;

    // Thread indices within the block
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Determine global indices
    int bx = blockIdx_x;
    int by = blockIdx_y;
    int i = bx * BX + tx;
    int j = by * BY + ty;

    // Shared memory tile with padding to avoid bank conflict
    __shared__ float tile[BY][BX + 1];

    // Boundary check
    if (i < n && j < n) {
        // Load data into shared memory
        tile[ty][tx] = a[j * n + i];
    }

    __syncthreads();

    // Transpose indices
    i = by * BY + tx;
    j = bx * BX + ty;

    // Check boundaries again before writing to global memory
    if (i < n && j < n) {
        b[j * n + i] = tile[tx][ty];
    }
}