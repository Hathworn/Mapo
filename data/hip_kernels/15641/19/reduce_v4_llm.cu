#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void warp_reduce(float* S, int tx) {
    // Optimize by using warp-synchronous primitives
    if (tx < 32) {
        volatile float* vsmem = S;
        vsmem[tx] += vsmem[tx + 32];
        vsmem[tx] += vsmem[tx + 16];
        vsmem[tx] += vsmem[tx + 8];
        vsmem[tx] += vsmem[tx + 4];
        vsmem[tx] += vsmem[tx + 2];
        vsmem[tx] += vsmem[tx + 1];
    }
}

__global__ void reduce_v4(float* in, float* out, int n) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BX = blockDim.x;
    int i = bx * (BX * 2) + tx;

    __shared__ float S[THEAD_MAX];

    // Preload data and reduce across threads in a block
    S[tx] = in[i] + in[i + BX];
    __syncthreads();

    for (int s = BX / 2; s > WARP_SIZE; s >>= 1) {
        if (tx < s) {
            S[tx] += S[tx + s];
        }
        __syncthreads();
    }
    
    // Final warp reduction
    if (tx < WARP_SIZE) {
        warp_reduce(S, tx);
    }
    
    // Write the result of this block to global memory
    if (tx == 0) {
        out[bx] = S[0];
    }
}