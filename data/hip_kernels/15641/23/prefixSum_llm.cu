#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prefixSum(float* arr, int step) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Efficient boundary check
    if (i >= step) {
        arr[i] += arr[i - step];
    }
}