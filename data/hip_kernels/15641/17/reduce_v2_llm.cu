#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce_v2(float* in, float* out, int n) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BX = blockDim.x; // same as THEAD_MAX
    int i  = bx * BX + tx;

    extern __shared__ float S[]; // Use dynamic shared memory

    S[tx] = (i < n) ? in[i] : 0;
    __syncthreads();

    // Use a loop unrolling technique for optimization
    for (int s = BX / 2; s > 32; s >>= 1) {
        if (tx < s) {
            S[tx] += S[tx + s];
        }
        __syncthreads();
    }

    if (tx < 32) {
        volatile float* Smem = S; // Using volatile to ensure no optimization issues
        Smem[tx] += Smem[tx + 32];
        Smem[tx] += Smem[tx + 16];
        Smem[tx] += Smem[tx + 8];
        Smem[tx] += Smem[tx + 4];
        Smem[tx] += Smem[tx + 2];
        Smem[tx] += Smem[tx + 1];
    }

    if (tx == 0) {
        out[bx] = S[0];
    }
}