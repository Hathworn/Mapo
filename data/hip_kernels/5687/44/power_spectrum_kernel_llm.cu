#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;

    // Launch sufficient threads to cover half_length, avoiding conditional inside the loop
    for (int idx = thread_id + 1; idx < half_length; idx += blockDim.x) {
        float2 val = reinterpret_cast<float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret);
    }

    // Handle special case with just one thread
    if (thread_id == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}