#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel( int32_t n, float old_num_frames, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    float cur_weight = *cur_tot_weight;

    float new_num_frames = old_num_frames + cur_weight;
    float prior_scale_change = 1.0f;

    // Calculate prior scale change only if max_count is non-zero
    if(max_count != 0) {
        float old_prior_scale = max(old_num_frames, (float)max_count) / max_count;
        float new_prior_scale = max(new_num_frames, (float)max_count) / max_count;
        prior_scale_change += new_prior_scale - old_prior_scale;
    }

    // Using a single loop to update quadratic terms
    int32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride = blockDim.x * gridDim.x;
    for (; index < n; index += stride) {
        int32_t diag_idx = ((index + 1) * (index + 2) / 2) - 1;
        quadratic[diag_idx] += prior_scale_change;
    }

    // Only one thread updates linear if it is the first thread of first block
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        linear[0] += prior_offset * prior_scale_change;
    }
}