#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Calculate global thread indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure indices are within bounds
    if (row < num_rows && col < num_cols) {
        float f = feats[row * ldf + col];
        feats_sq[row * lds + col] = f * f; // Square matrix element
    }
}