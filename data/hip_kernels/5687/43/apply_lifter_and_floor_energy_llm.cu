#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void apply_lifter_and_floor_energy( int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    float *feats = features + frame * ldf;

    // Use shared memory for lifter_coeffs for faster access
    extern __shared__ float shared_lifter_coeffs[];
    if (thread_id < num_cols) {
        shared_lifter_coeffs[thread_id] = lifter_coeffs[thread_id];
    }
    __syncthreads();

    // Apply lifter coefficients concurrently
    if (cepstral_lifter != 0.0f) {
        for (int c = thread_id; c < num_cols; c += blockDim.x) {
            float lift = shared_lifter_coeffs[c];
            feats[c] *= lift;
        }
    }

    // Use __syncthreads before accessing shared data
    __syncthreads();

    // Thread 0 for each frame will handle energy operations
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = logf(energy_floor);

        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        feats[0] = energy;
    }
}