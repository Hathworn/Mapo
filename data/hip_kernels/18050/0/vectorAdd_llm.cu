#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*-----This is a vector addition--*/
/*---- @ Cuda/c ------*/
/*---- __NS__Bologna__2020__*/

__global__ void vectorAdd(int* a, int* b, int* c, int n) {
    // Calculate index thread
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize by using shared memory to reduce global memory access
    extern __shared__ int shared[];
    if (tid < n) {
        shared[threadIdx.x] = a[tid] + b[tid];
        __syncthreads();
        c[tid] = shared[threadIdx.x];
    }
}