#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized matrix multiplication kernel
__global__ void matriMult(int* m, int* n, int* p, int size) {
    // Calculate Row and Column indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int column = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Boundary check
    if (row < size && column < size) {
        int p_sum = 0;
        // Loop unrolling for optimization
        for (int i = 0; i < size; i++) {
            p_sum += m[row * size + i] * n[i * size + column];
        }
        // Store the result
        p[row * size + column] = p_sum;
    }
}