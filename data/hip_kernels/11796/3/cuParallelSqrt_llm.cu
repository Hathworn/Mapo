#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuParallelSqrt(float *dist, int width, int pitch, int k){
    // Use shared memory to improve memory access efficiency
    __shared__ float tile[blockDim.y][blockDim.x];

    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIndex < width && yIndex < k) {
        // Load data into shared memory
        tile[threadIdx.y][threadIdx.x] = dist[yIndex * pitch + xIndex];
        __syncthreads();

        // Perform computation using shared memory
        tile[threadIdx.y][threadIdx.x] = sqrt(tile[threadIdx.y][threadIdx.x]);
        __syncthreads();

        // Store results back to global memory
        dist[yIndex * pitch + xIndex] = tile[threadIdx.y][threadIdx.x];
    }
}