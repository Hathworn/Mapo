#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuInsertionSort(float *dist, int dist_pitch, int *ind, int ind_pitch, int width, int height, int k) {

    // Using shared memory for faster access
    extern __shared__ float shared_mem[];
    int* shared_ind = (int*)&shared_mem[k * blockDim.x];
  
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (xIndex < width) {

        float* p_dist = dist + xIndex;
        int* p_ind = ind + xIndex;
    
        // Load data into shared memory
        for (int idx = threadIdx.y; idx < k; idx += blockDim.y) {
            shared_mem[threadIdx.x + idx * blockDim.x] = p_dist[idx * dist_pitch];
            shared_ind[threadIdx.x + idx * blockDim.x] = p_ind[idx * ind_pitch];
        }
        __syncthreads();

        // Perform insertion sort in shared memory
        float max_dist = shared_mem[threadIdx.x];
        shared_ind[threadIdx.x] = 1;

        for (int l = 1; l < k; ++l) {
            float curr_dist = shared_mem[threadIdx.x + l * blockDim.x];
            if (curr_dist < max_dist) {
                int i = l - 1;
                for (int a = 0; a < l - 1; ++a) {
                    if (shared_mem[threadIdx.x + a * blockDim.x] > curr_dist) {
                        i = a;
                        break;
                    }
                }
                for (int j = l; j > i; --j) {
                    shared_mem[threadIdx.x + j * blockDim.x] = shared_mem[threadIdx.x + (j - 1) * blockDim.x];
                    shared_ind[threadIdx.x + j * blockDim.x] = shared_ind[threadIdx.x + (j - 1) * blockDim.x];
                }
                shared_mem[threadIdx.x + i * blockDim.x] = curr_dist;
                shared_ind[threadIdx.x + i * blockDim.x] = l + 1;
            } else {
                shared_ind[threadIdx.x + l * blockDim.x] = l + 1;
                max_dist = curr_dist;
            }
        }

        // Additional sorting for the kth elements
        int max_row = (k - 1) * dist_pitch;
        for (int l = k; l < height; ++l) {
            float curr_dist = p_dist[l * dist_pitch];
            if (curr_dist < max_dist) {
                int i = k - 1;
                for (int a = 0; a < k - 1; ++a) {
                    if (shared_mem[threadIdx.x + a * blockDim.x] > curr_dist) {
                        i = a;
                        break;
                    }
                }
                for (int j = k - 1; j > i; --j) {
                    shared_mem[threadIdx.x + j * blockDim.x] = shared_mem[threadIdx.x + (j - 1) * blockDim.x];
                    shared_ind[threadIdx.x + j * blockDim.x] = shared_ind[threadIdx.x + (j - 1) * blockDim.x];
                }
                shared_mem[threadIdx.x + i * blockDim.x] = curr_dist;
                shared_ind[threadIdx.x + i * blockDim.x] = l + 1;
                max_dist = shared_mem[threadIdx.x + max_row];
            }
        }
        
        // Copy back sorted data from shared memory to global memory
        for (int idx = threadIdx.y; idx < k; idx += blockDim.y) {
            p_dist[idx * dist_pitch] = shared_mem[threadIdx.x + idx * blockDim.x];
            p_ind[idx * ind_pitch] = shared_ind[threadIdx.x + idx * blockDim.x];
        }
    }
}