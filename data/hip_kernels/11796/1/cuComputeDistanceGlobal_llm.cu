#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_DIM 16  // Assuming BLOCK_DIM is defined

__global__ void cuComputeDistanceGlobal(float* A, int wA, int pA, float* B, int wB, int pB, int dim, float* AB){
    extern __shared__ float shared_mem[]; // Use shared memory efficiently 
    float* shared_A = shared_mem;
    float* shared_B = shared_mem + BLOCK_DIM * BLOCK_DIM;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float ssd = 0;

    int begin_A = BLOCK_DIM * blockIdx.y;
    int begin_B = BLOCK_DIM * blockIdx.x;
    int step_A = BLOCK_DIM * pA;
    int step_B = BLOCK_DIM * pB;
    int end_A = begin_A + (dim - 1) * pA;

    int cond0 = (begin_A + tx < wA);
    int cond1 = (begin_B + tx < wB);
    int cond2 = (begin_A + ty < wA);

    for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {
        if (a / pA + ty < dim) {
            shared_A[ty * BLOCK_DIM + tx] = (cond0) ? A[a + pA * ty + tx] : 0;
            shared_B[ty * BLOCK_DIM + tx] = (cond1) ? B[b + pB * ty + tx] : 0;  
        } else {
            shared_A[ty * BLOCK_DIM + tx] = 0;
            shared_B[ty * BLOCK_DIM + tx] = 0;
        }

        __syncthreads();

        if (cond2 && cond1) {
            for (int k = 0; k < BLOCK_DIM; ++k) {
                float tmp = shared_A[k * BLOCK_DIM + ty] - shared_B[k * BLOCK_DIM + tx];
                ssd += tmp * tmp;
            }
        }

        __syncthreads();
    }

    if (cond2 && cond1)
        AB[(begin_A + ty) * pB + begin_B + tx] = ssd;
}