#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy(float a, float* x, float* y, int n) {
    // Calculate global index for concurrent execution
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Perform axpy operation if index is within bounds
    if (idx < n) {
        y[idx] = a * x[idx];
    }
}