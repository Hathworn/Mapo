#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "MyClass.h"

class MyCudaClass : public MyClass {
    public:
        using Base = MyClass;

        double *devParam;
        const size_t dubSize = sizeof(double);

        int nCudaBlocks = 1;
        int nCudaThreadsPerBlock = 1;

        MyCudaClass(){
            hipMalloc((void **)&devParam, dubSize);
        }

        ~MyCudaClass(){
            hipFree(devParam);
        }

        void set_param(double in){
            Base::set_param(in);
            hipMemcpy(devParam, &(Base::hostParam), dubSize, hipMemcpyHostToDevice);
        }

        double do_it_on_device(){
            double *devOut, out;
            hipMalloc((void **)&devOut, dubSize);
            devKernel<<< nCudaBlocks, nCudaThreadsPerBlock >>>(devParam, devOut);
            hipMemcpy(&out, devOut, dubSize, hipMemcpyDeviceToHost);
            return out;
        }

        __global__ static void devKernel(double *param, double *ans){
            // Cuda implementation
            std::printf("Inside devKernel: ");
            *ans = *param + 3.14;
        }

};