#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 2560
#define M 512
#define BLOCK_SIZE (N/M)
#define RADIUS 5

__global__ void add(double *a, double *b, double *c, int n){
    // Utilize shared memory for better memory access performance
    extern __shared__ double shared_a[];

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int localIdx = threadIdx.x;

    if (idx < n) {
        // Load data into shared memory
        shared_a[localIdx] = a[idx];
        __syncthreads(); // Ensure all threads have loaded their data

        // Perform computation
        c[idx] = shared_a[localIdx] + b[idx];
    }
}