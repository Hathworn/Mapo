#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    float *feats = features + frame * ldf;

    // Apply lifter coefficients using shared memory for efficiency.
    if (cepstral_lifter != 0.0f) {
        // Use shared memory for lifter coefficients
        __shared__ float shared_lifter_coeffs[1024]; // Adjust size as needed
        if (thread_id < num_cols) {
            shared_lifter_coeffs[thread_id] = lifter_coeffs[thread_id];
        }
        __syncthreads();
        
        for (int c = thread_id; c < num_cols; c += blockDim.x) {
            float lift = shared_lifter_coeffs[c];
            float f = feats[c];
            feats[c] = f * lift;
        }
    }

    // Thread 0 for each frame will apply energy
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = logf(energy_floor); // use logf for float precision

        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        feats[0] = energy;
    }
}