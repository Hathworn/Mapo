#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    extern __shared__ float shared_A_in[];  // Use shared memory for input

    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;

    // Load input data into shared memory
    for (int idx = thread_id; idx < row_length; idx += blockDim.x) {
        shared_A_in[idx] = Ar[idx];
    }
    __syncthreads();  // Synchronize threads

    // Main computation
    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {
        if (idx == 0) continue;  // Skip special case

        float2 val = reinterpret_cast<float2 *>(shared_A_in)[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret);
    }

    // Handle special case
    if (thread_id == 0) {
        float real = shared_A_in[0];
        float im = shared_A_in[row_length];

        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}