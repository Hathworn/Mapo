#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float logarithmic_mapping(float k, float q, float val_pixel, float maxLum) {
    return (log10f(1.0 + q * val_pixel))/(log10f(1.0 + k * maxLum));
}

__device__ float rgb2Lum(float B, float G, float R) {
    return B * 0.0722 + G * 0.7152 + R * 0.2126;
}

__global__ void log_tonemap_kernel(float* imageIn, float* imageOut, int width, int height, int channels, float k, float q, float* max) {
    int Row = blockDim.y * blockIdx.y + threadIdx.y;
    int Col = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure row and column are within image boundaries
    if (Row < height && Col < width) {
        // Load pixel components
        float B = imageIn[(Row * width + Col) * channels + BLUE];
        float G = imageIn[(Row * width + Col) * channels + GREEN];
        float R = imageIn[(Row * width + Col) * channels + RED];

        // Calculate luminance and apply logarithmic tone mapping and scaling
        float L = rgb2Lum(B, G, R);
        float nL = logarithmic_mapping(k, q, L, *max);
        float scale = (L > 0) ? (nL / L) : 0.0f; // Avoid divide by zero

        // Store results back to the output image
        imageOut[(Row * width + Col) * channels + BLUE] = B * scale;
        imageOut[(Row * width + Col) * channels + GREEN] = G * scale;
        imageOut[(Row * width + Col) * channels + RED] = R * scale;
    }
}