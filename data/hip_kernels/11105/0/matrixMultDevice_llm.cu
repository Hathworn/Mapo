#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TAM 2

__global__ void matrixMultDevice(float* d_A, float* d_B, float* d_C, int width) {
    // Use shared memory to optimize memory access
    __shared__ float shared_A[TAM][TAM];
    __shared__ float shared_B[TAM][TAM];

    int Row = blockDim.y * blockIdx.y + threadIdx.y;
    int Col = blockDim.x * blockIdx.x + threadIdx.x;

    float ans = 0.0;

    // Loop over the tiles of the input arrays
    for (int tileIdx = 0; tileIdx < gridDim.x; ++tileIdx) {
        // Collaboratively load tiles into shared memory
        if (Row < width && tileIdx * TAM + threadIdx.x < width)
            shared_A[threadIdx.y][threadIdx.x] = d_A[Row * width + tileIdx * TAM + threadIdx.x];
        else
            shared_A[threadIdx.y][threadIdx.x] = 0.0;

        if (Col < width && tileIdx * TAM + threadIdx.y < width)
            shared_B[threadIdx.y][threadIdx.x] = d_B[(tileIdx * TAM + threadIdx.y) * width + Col];
        else
            shared_B[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        // Multiply the two matrices together
        for (int k = 0; k < TAM; ++k) {
            ans += shared_A[threadIdx.y][k] * shared_B[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Write the block sub-matrix to global memory
    if (Row < width && Col < width) {
        d_C[Row * width + Col] = ans;
    }
}