#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMultDevice(float* d_A, float* d_B, float* d_C, int width) {
    int Row = blockDim.y * blockIdx.y + threadIdx.y;
    int Col = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory for tile optimization
    __shared__ float shared_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float shared_B[BLOCK_SIZE][BLOCK_SIZE];

    float ans = 0.0;

    // Iterate over tiles
    for (int i = 0; i < width / BLOCK_SIZE; ++i) {
        // Load tiles into shared memory
        shared_A[threadIdx.y][threadIdx.x] = d_A[Row * width + (i * BLOCK_SIZE + threadIdx.x)];
        shared_B[threadIdx.y][threadIdx.x] = d_B[(i * BLOCK_SIZE + threadIdx.y) * width + Col];

        // Synchronize threads to ensure all data is loaded
        __syncthreads();

        // Matrix multiplication on the block
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            ans += shared_A[threadIdx.y][k] * shared_B[k][threadIdx.x];
        }

        // Synchronize before loading new tiles
        __syncthreads();
    }

    if (Row < width && Col < width) {
        d_C[Row * width + Col] = ans;
    }
}