#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float* __restrict__ d_A, float* __restrict__ d_B, float* __restrict__ d_C) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < TAM) {
        // Use registers to hold the values, minimizing global memory access
        float a_val = d_A[i];
        float b_val = d_B[i];
        d_C[i] = a_val + b_val;
    }
}