#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access by reducing redundant calculations
__global__ void rgb2gray(unsigned char* d_Pin, unsigned char* d_Pout, int width, int height) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if ((Row < height) && (Col < width)) {
        int idx = (Row * width + Col) * 3; // Calculate once
        unsigned char blue = d_Pin[idx + BLUE];
        unsigned char green = d_Pin[idx + GREEN];
        unsigned char red = d_Pin[idx + RED];
        
        d_Pout[Row * width + Col] = blue * 0.114 + green * 0.587 + red * 0.299;
    }
}