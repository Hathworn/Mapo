#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float adaptive_logarithmic_mapping(float lw_max, float ld_max, float lw, float b)
{
    float ld = (ld_max / (100.0f * log10f(1.0f + lw_max))) *
               (logf(1.0f + lw) / (logf(2.0f + 8.0f * powf((lw / lw_max), (logf(b) / logf(0.5f))))));
    return ld;
}

__device__ float logarithmic_mapping(float k, float q, float val_pixel, float maxLum)
{
    return (log10f(1.0f + q * val_pixel)) / (log10f(1.0f + k * maxLum));
}

__device__ float rgb2Lum(float B, float G, float R)
{
    return B * 0.0722f + G * 0.7152f + R * 0.2126f;
}

__global__ void adaptive_log_tonemap_kernel(float* imageIn, float* imageOut, int width, int height, int channels, float b, float ld_max, float* max)
{
    int Row = blockDim.y * blockIdx.y + threadIdx.y;
    int Col = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if within image boundaries
    if (Row < height && Col < width) {
        // Calculate linear index for 3-channel image data
        int idx = (Row * width + Col) * 3;

        // Read input image values
        float B = imageIn[idx + BLUE];
        float G = imageIn[idx + GREEN];
        float R = imageIn[idx + RED];

        // Calculate luminance
        float L = rgb2Lum(B, G, R);

        // Perform adaptive logarithmic mapping
        float nL = adaptive_logarithmic_mapping(*max, ld_max, L, b);

        // Calculate scale and apply to each channel
        float scale = nL / L;
        imageOut[idx + BLUE] = B * scale;
        imageOut[idx + GREEN] = G * scale;
        imageOut[idx + RED] = R * scale;
    }
}