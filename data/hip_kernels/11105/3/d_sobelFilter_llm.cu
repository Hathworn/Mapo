#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void d_sobelFilter(unsigned char* imageIn, unsigned char* imageOut, int width, int height, int maskWidth, char* M) {
    int Row = blockDim.y * blockIdx.y + threadIdx.y;
    int Col = blockDim.x * blockIdx.x + threadIdx.x;

    if(Row < height && Col < width) {
        int res = 0;
        for(int i = 0; i < maskWidth; i++) {
            for(int j = 0; j < maskWidth; j++) {
                int nrow = Row + i - (maskWidth / 2);
                int ncol = Col + j - (maskWidth / 2);
                // Check image boundary before accessing memory
                if(nrow >= 0 && nrow < height && ncol >= 0 && ncol < width) {
                    res += imageIn[nrow * width + ncol] * M[i * maskWidth + j];
                }
            }
        }
        // Clamp result to valid range
        imageOut[Row * width + Col] = (unsigned char)min(max(res, 0), 255);
    }
}