#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLUE 0
#define GREEN 1
#define RED 2

__device__ float gamma_correction(float f_stop, float gamma, float val) {
    return powf((val * powf(2.0, f_stop)), (1.0 / gamma));
}

__device__ float rgb2Lum(float B, float G, float R) {
    return B * 0.0722 + G * 0.7152 + R * 0.2126;
}

__global__ void gamma_tonemap_kernel(float* imageIn, float* imageOut, int width, int height, int channels, float f_stop, float gamma) {
    int Row = blockDim.y * blockIdx.y + threadIdx.y;
    int Col = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure processing within image bounds
    if (Row < height && Col < width) {
        int index = (Row * width + Col) * channels; // Precompute index for readability and reuse
        float B = imageIn[index + BLUE];
        float G = imageIn[index + GREEN];
        float R = imageIn[index + RED];

        float L = rgb2Lum(B, G, R);
        float nL = gamma_correction(f_stop, gamma, L);
        float scale = nL / L;

        imageOut[index + BLUE] = B * scale;
        imageOut[index + GREEN] = G * scale;
        imageOut[index + RED] = R * scale;
    }
}