#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rgb2gray(unsigned char* d_Pin, unsigned char* d_Pout, int width, int height) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = (Row * width + Col) * 3;  // Precompute index for performance optimization

    if ((Row < height) && (Col < width)) {
        unsigned char blue = d_Pin[idx + BLUE];
        unsigned char green = d_Pin[idx + GREEN];
        unsigned char red = d_Pin[idx + RED];
        d_Pout[Row * width + Col] = blue * 0.114 + green * 0.587 + red * 0.299;
    }
}