#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float rgb2Lum(float B, float G, float R)
{
    return B * 0.0722 + G * 0.7152 + R * 0.2126;
}

__global__ void find_maximum_kernel(float *array, float *max, int *mutex, unsigned int n)
{
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int stride = gridDim.x * blockDim.x;
    
    extern __shared__ float cache[];

    float temp = -1.0f;
    
    // Use unrolling to improve data fetch efficiency
    while (index * 3 + 2 < n) {
        float B, G, R, L;
        B = array[index * 3 + BLUE];
        G = array[index * 3 + GREEN];
        R = array[index * 3 + RED];
        L = rgb2Lum(B, G, R);

        temp = fmaxf(temp, L);

        index += stride;
    }

    cache[threadIdx.x] = temp;
    __syncthreads();

    // Improved reduction with warp level optimizations (assumes blockDim.x is a power of 2)
    for (unsigned int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (threadIdx.x < i) {
            cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x + i]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        while (atomicCAS(mutex, 0, 1) != 0);  // lock
        *max = fmaxf(*max, cache[0]);
        atomicExch(mutex, 0);  // unlock
    }
}