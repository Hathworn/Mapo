#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__global__ void matrixMultTiled(float* d_A, float* d_B, float* d_C, int width) {
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Identify the row and column of the d_P element to work on
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float ans = 0;
    // Loop over the d_M and d_N tiles required to compute d_P element
    for(int m = 0; m < (width + TILE_WIDTH - 1) / TILE_WIDTH; m++) {
        // Load tiles into shared memory with boundary check
        if(m * TILE_WIDTH + tx < width && Row < width)
            ds_A[ty][tx] = d_A[Row * width + m * TILE_WIDTH + tx];
        else
            ds_A[ty][tx] = 0.0f;

        if(m * TILE_WIDTH + ty < width && Col < width)
            ds_B[ty][tx] = d_B[(m * TILE_WIDTH + ty) * width + Col];
        else
            ds_B[ty][tx] = 0.0f;

        __syncthreads();

        // Perform computation
        for(int k = 0; k < TILE_WIDTH; k++) {
            ans += ds_A[ty][k] * ds_B[k][tx];
        }
        __syncthreads();
    }
    
    // Write result to global memory
    if(Row < width && Col < width)
        d_C[Row * width + Col] = ans;
}