#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rgb2gray(unsigned char* d_Pin, unsigned char* d_Pout, int width, int height) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for pixel RGB values to reduce global memory access
    __shared__ unsigned char shared_d_Pin[1024 * 3];  // adjust size according to block size

    if ((Row < height) && (Col < width)) {
        // Calculate linear thread index within the block
        int threadIdx_linear = threadIdx.y * blockDim.x + threadIdx.x;

        // Load pixel values into shared memory
        shared_d_Pin[threadIdx_linear * 3 + 0] = d_Pin[(Row * width + Col) * 3 + 0];
        shared_d_Pin[threadIdx_linear * 3 + 1] = d_Pin[(Row * width + Col) * 3 + 1];
        shared_d_Pin[threadIdx_linear * 3 + 2] = d_Pin[(Row * width + Col) * 3 + 2];

        __syncthreads();  // Ensure all threads have loaded their pixels

        // Use shared memory for computation
        d_Pout[Row * width + Col] = shared_d_Pin[threadIdx_linear * 3 + BLUE] * 0.114 +
                                    shared_d_Pin[threadIdx_linear * 3 + GREEN] * 0.587 +
                                    shared_d_Pin[threadIdx_linear * 3 + RED] * 0.299;
    }
}