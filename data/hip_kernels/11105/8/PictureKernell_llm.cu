#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PictureKernell(unsigned char* d_Pin, unsigned char* d_Pout, int n, int m) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure both Row and Col are within bounds
    if (Row < m && Col < n) {
        int index = Row * n + Col; // Calculate linear index once
        d_Pout[index] = 2 * d_Pin[index]; // Access memory directly with computed index
    }
}