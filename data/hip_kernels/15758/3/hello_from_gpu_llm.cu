#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized Kernel Function
__global__ void hello_from_gpu()
{
    // Utilize block and thread indices for scalability
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Condition to limit excessive print in larger configurations
    if(idx == 0)
    {
        printf("Hello World from the GPU!\n");
    }
}