#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_from_gpu() 
{
    // Cache block and thread indices in registers for faster access
    const int b = blockIdx.x;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    // Use a single printf statement to optimize performance
    printf("Hello World from block-%d and thread-(%d, %d)!\n", b, tx, ty);
}