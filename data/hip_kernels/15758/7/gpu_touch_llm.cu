#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_touch(uint64_t *x, const size_t size)
{
    // Utilize grid stride loop for better efficiency and avoid warp divergence
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x)
    {
        x[i] = 0;
    }
}