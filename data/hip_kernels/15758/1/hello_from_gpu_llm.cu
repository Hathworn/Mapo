#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_from_gpu()
{
    // Calculate unique thread index for better performance understanding
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello World from global index %d!\n", tid);
}