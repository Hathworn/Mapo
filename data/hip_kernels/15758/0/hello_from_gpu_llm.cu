#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_from_gpu()
{
    // Use thread index to distribute work across threads
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        printf("Hello World from the GPU!\n");
    }
}