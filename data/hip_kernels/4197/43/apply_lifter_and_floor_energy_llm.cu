#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;
    
    if (frame >= num_frames) return;  // Ensure no excess computation if grid is larger

    float *feats = features + frame * ldf;

    // Optimization: Using shared memory for lifter coefficients if possible
    __shared__ float shared_lifter_coeffs[256];  // Assumes max of 256 columns, adjust if necessary
    if (thread_id < num_cols) {
        shared_lifter_coeffs[thread_id] = lifter_coeffs[thread_id];
    }
    __syncthreads();

    // Apply lifter coefficients using shared memory
    if (cepstral_lifter != 0.0f) {
        for (int c = thread_id; c < num_cols; c += blockDim.x) {
            float lift = shared_lifter_coeffs[c];
            float f = feats[c];
            feats[c] = f * lift;
        }
    }

    // Thread 0 for each frame will apply energy
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = log(energy_floor);

        // Apply energy flooring if necessary
        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        feats[0] = energy;
    }
}