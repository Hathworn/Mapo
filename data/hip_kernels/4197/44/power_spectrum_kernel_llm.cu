#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;

    // Process all indices in parallel avoiding index zero
    for (int idx = thread_id + 1; idx < half_length; idx += blockDim.x) {
        float2 val = reinterpret_cast<float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret);
    }

    // Synchronize threads before processing special cases
    __syncthreads();
    
    // Handle special cases with thread 0
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];
        Aw[0] = use_power ? real * real : fabsf(real);
        Aw[half_length] = use_power ? im * im : fabsf(im);
    }
}