#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    // Get global thread index
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate half_length outside of the loop
    int half_length = row_length / 2;

    // Early exit for thread_id >= half_length
    if (thread_id >= half_length) return;

    const float *Ar = A_in + blockIdx.x * ldi;
    float *Aw = A_out + blockIdx.x * ldo;

    // Skip processing if thread_id == 0, handled separately
    if (thread_id != 0) {
        float2 val = reinterpret_cast<const float2 *>(Ar)[thread_id];
        float ret = val.x * val.x + val.y * val.y;
        Aw[thread_id] = use_power ? ret : sqrtf(ret);
    }

    // Handle special case for thread_id == 0, executed by the first thread of each block
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length]; // cufft puts this at the end

        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}