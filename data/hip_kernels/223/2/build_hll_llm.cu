#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void build_hll(int n, unsigned int *in, unsigned int *out) {
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread is within bounds
    if (offset >= n) return;

    // Extract the parts
    unsigned int val = in[offset];
    int bucket = val >> HLL_BUCKET_WIDTH;
    int pos = val & ((1 << HLL_BUCKET_WIDTH) - 1);

    // Atomically update the maximum position
    atomicMax(&out[bucket], pos);
}