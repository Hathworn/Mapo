#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpy(double *y, double *a, double *x, int m, int n) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];

    // Use a more compact thread mapping for better memory coalescing
    int index = bid * n + tid;

    if (bid < m && tid < n) {
        dots_s[tid] = a[index] * x[tid];
        __syncthreads();

        // Utilize parallel reduction for accumulation
        for (int stride = n / 2; stride > 0; stride >>= 1) {
            if (tid < stride) {
                dots_s[tid] += dots_s[tid + stride];
            }
            __syncthreads();
        }

        // Only the first thread writes the result
        if (tid == 0) {
            y[bid] = dots_s[0];
        }
    }
}