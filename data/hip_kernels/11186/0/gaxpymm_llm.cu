#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpymm(double *y, double *a, double *b, int m, int n, int p) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];

    if (bid < m) {
        if (tid < n) {
            // Initialize shared memory partial results
            for (int c = 0; c < p; c++) {
                dots_s[tid*p + c] = a[bid*n + tid] * b[tid*p + c];
            }
        }
        __syncthreads();

        // Parallel reduction to accumulate results into dots_s[0] for each column
        for (int stride = n/2; stride > 0; stride /= 2) {
            if (tid < stride) {
                for (int c = 0; c < p; c++) {
                    dots_s[tid*p + c] += dots_s[(tid + stride)*p + c];
                }
            }
            __syncthreads();
        }

        // Thread 0 writes the result
        if (tid == 0) {
            for (int c = 0; c < p; c++) {
                y[bid*p + c] = dots_s[c];
            }
        }
    }
}