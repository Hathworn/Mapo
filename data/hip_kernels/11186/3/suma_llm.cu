#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void suma(int a, int b, int *c) {
    // Use thread ID to ensure each thread performs distinct operations (if needed, considering only 1 thread here)
    int tid = threadIdx.x;
    if (tid == 0) { // Limit execution to the first thread
        *c = a + b;
    }
}