#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__global__ void MatrixMul(float *Md, float *Nd, float *Pd, const int WIDTH) {
    // Calculate thread id
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < WIDTH && row < WIDTH) { // Ensure within bounds
        float Pvalue = 0; // Accumulate results
        for (int k = 0; k < WIDTH; k++) {
            Pvalue += Md[row * WIDTH + k] * Nd[k * WIDTH + col];
        }
        Pd[row * WIDTH + col] = Pvalue; // Write result
    }
}