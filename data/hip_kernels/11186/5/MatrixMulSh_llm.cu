#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulSh(float *Md, float *Nd, float *Pd, const int WIDTH)
{
    // Use shared memory to load tiles of Md and Nd
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // Calculate thread's global row and column index
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

    float Pvalue = 0; // Accumulate product results

    // Loop over the Md and Nd tiles required to compute Pd element
    for (int m = 0; m < WIDTH / TILE_WIDTH; ++m)
    {
        // Load Md and Nd elements into shared memory
        Mds[threadIdx.y][threadIdx.x] = Md[row * WIDTH + (m * TILE_WIDTH + threadIdx.x)];
        Nds[threadIdx.y][threadIdx.x] = Nd[(m * TILE_WIDTH + threadIdx.y) * WIDTH + col];
        __syncthreads(); // Synchronize to ensure tiles are loaded

        // Multiply the tiles together
        for (int k = 0; k < TILE_WIDTH; ++k)
        {
            Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
        }
        __syncthreads(); // Synchronize to ensure computation is complete before loading new tiles
    }

    // Write the computed value to Pd
    Pd[row * WIDTH + col] = Pvalue;
}