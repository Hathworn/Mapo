#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinput(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute 1D block index for unique indexing
    int global_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    // Calculate x and y coordinates based on the global index
    int yy = global_idx / output_w;
    int xx = global_idx % output_w;

    // Ensure indices are within bounds
    if (yy < output_h && xx < output_w) {
        // Calculate the offsets for the gradOutput and gradInput pointers
        int o = blockIdx.x;
        int i = blockIdx.x;
        int k = blockIdx.x % input_n;

        gradOutput = gradOutput + o * output_w * output_h;
        gradInput = gradInput + i * input_w * input_h;

        // Fetch the corresponding weight
        float the_weight = weight[k];

        // Compute gradInput contribution for the current thread
        float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float z = *ptr_gradOutput * the_weight;
        
        for (int ky = 0; ky < kH; ky++) {
            for (int kx = 0; kx < kW; kx++) {
                ptr_gradInput[kx] += z;
            }
            ptr_gradInput += input_w;
        }
    }
}