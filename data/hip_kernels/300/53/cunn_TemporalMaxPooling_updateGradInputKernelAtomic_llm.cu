#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Compute base indices for gradInput, gradOutput, and indices using shared memory
    extern __shared__ float sdata[];
    int tid = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;

    if (tid < output_w) { // Ensuring time index is within bounds
        float *gradInput_base = gradInput + blockIdx.x * input_w * input_n;
        float *gradOutput_base = gradOutput + blockIdx.x * output_w * input_n;
        float *indices_base = indices + blockIdx.x * output_w * input_n;

        gradInput_base += tid * input_n * dW;
        gradOutput_base += tid * input_n;
        indices_base += tid * input_n;

        // For all features
        for (int feat = 0; feat < input_n; ++feat) {
            int idx = static_cast<int>(indices_base[feat]) * input_n + feat;
            atomicAdd(&gradInput_base[idx], gradOutput_base[feat]);
        }
    }
}