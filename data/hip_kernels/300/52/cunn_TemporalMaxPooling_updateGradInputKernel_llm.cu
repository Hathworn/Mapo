#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Improved memory access patterns for coalescing
    int global_index = threadIdx.x + blockIdx.y * blockDim.x;
    if (global_index < output_w) {
        int batch_offset = blockIdx.x * input_w * input_n;
        gradInput += batch_offset;
        gradOutput += batch_offset / input_w * output_w;
        indices += batch_offset / input_w * output_w;
        
        // Align pointers for this thread's output position
        gradInput += global_index * dW * input_n;
        gradOutput += global_index * input_n;
        indices += global_index * input_n;
        
        // Loop unrolling for potential performance gain
        for (int feat = 0; feat < input_n; ++feat) {
            int input_index = static_cast<int>(indices[feat]) * input_n + feat;
            gradInput[input_index] += gradOutput[feat];
        }
    }
}