#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w)
{
    // Calculate thread coordinates
    int o = blockIdx.x;
    int i = o;
    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Offset pointers based on plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // Loop over all output elements
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        int y_start = static_cast<int>(yy * input_h / static_cast<float>(output_h));

        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            int x_start = static_cast<int>(xx * input_w / static_cast<float>(output_w));

            // Pointers to current gradOutput and indices
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float *ptr_ind_x = indices_x + yy * output_w + xx;
            float *ptr_ind_y = indices_y + yy * output_w + xx;
            float z = *ptr_gradOutput;

            // Argument max coordinates with bounds check
            int argmax_x = (*ptr_ind_x) - 1;
            int argmax_y = (*ptr_ind_y) - 1;

            // Ensure indexes are within bounds
            if (argmax_x >= 0 && argmax_x < input_w && argmax_y >= 0 && argmax_y < input_h) {
                gradInput[(y_start + argmax_y) * input_w + (x_start + argmax_x)] += z;
            }
        }
    }
}