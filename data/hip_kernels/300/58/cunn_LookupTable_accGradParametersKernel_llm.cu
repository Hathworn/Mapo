#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_LookupTable_accGradParametersKernel( float *input, float *indices, float *gradOutput, float *gradWeight, float *count, float defaultScale, long numel, long stride, int paddingValue) {

    int idx = blockIdx.x * 4 + threadIdx.y;

    // Number of values processed by each thread (grain size)
    const int SZ = 4;

    if (idx < numel
        && (idx == 0 || input[idx] != input[idx - 1])
        && input[idx] != paddingValue) {
        do {
            const int startFeature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
            const int weightRow = ((int) input[idx] - 1) * stride;
            const int gradOutputRow = ((int) indices[idx] - 1) * stride;
            const float scale = count ? defaultScale / count[idx] : defaultScale;

            float gradient[SZ];
            float weight[SZ];

            // Use thread-level parallelism within a warp
            for (int ii = threadIdx.x; ii < SZ; ii += WARP_SIZE) {
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride) {
                    gradient[ii] = gradOutput[gradOutputRow + featureDim];
                    weight[ii] = gradWeight[weightRow + featureDim];
                }
            }

            // Reduce memory transactions by directly updating gradWeight
            for (int ii = threadIdx.x; ii < SZ; ii += WARP_SIZE) {
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride) {
                    gradWeight[weightRow + featureDim] += gradient[ii] * scale;
                }
            }

            idx++;
        } while (idx < numel && input[idx] == input[idx - 1]);
    }
}