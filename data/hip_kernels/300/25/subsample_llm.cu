#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW) {
    // Calculate output dimensions and offsets based on thread/block ID
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int yy_start = blockIdx.y * blockDim.y + threadIdx.y;

    // Select input/output plane and compute weights/bias
    output += o * output_w * output_h;
    input += i * input_w * input_h;
    float the_weight = weight[k];
    float the_bias = bias[k];

    // Process each output pixel
    for (int yy = yy_start; yy < output_h; yy += blockDim.y * gridDim.y) {
        for (int xx = xx_start; xx < output_w; xx += blockDim.x) {
            float *ptr_input = input + yy * dH * input_w + xx * dW;
            float *ptr_output = output + yy * output_w + xx;
            float sum = 0;
            
            // Compute the mean of the input image using shared memory
            for (int ky = 0; ky < kH; ++ky) {
                for (int kx = 0; kx < kW; ++kx)
                    sum += ptr_input[kx];
                ptr_input += input_w; // next input line
            }
            
            // Update output using precomputed weight and bias
            *ptr_output = the_weight * sum + the_bias;
        }
    }
}