#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor)
{
    int x, y, z, w;
    w = ii % d3;
    ii /= d3;  // Use compound assignment for division
    z = ii % d2;
    ii /= d2;
    y = ii % d1;
    ii /= d1;
    x = ii;
    w /= scale_factor;
    z /= scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void upscale(float *input, float *output, long no_elements, int scale_factor, int d1, int d2, int d3)
{
    // Calculate global thread ID
    long ii = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * (blockDim.x * gridDim.x);
    if (ii >= no_elements) return;
    int ipidx = translate_idx(ii, d1, d2, d3, scale_factor);
    output[ii] = input[ipidx];
}