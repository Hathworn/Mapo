#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 8
#define NBLOCKS 4
#define NTHREADS (N/NBLOCKS)

__global__ void sum(int* in, int* out) {
    extern __shared__ int shared[];
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int bdim = blockDim.x;
    
    shared[tid] = in[bid * bdim + tid]; // Load input into shared memory
    __syncthreads();
    
    // Use a loop to perform the reduction
    for (int stride = bdim / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            shared[tid] += shared[tid + stride];
        }
        __syncthreads();
    }
    
    // Store the result in the output array
    if (tid == 0) {
        out[bid] = shared[0];
    }
}