#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int* a, int* b, int* c) {
    // Use threadIdx.x and blockDim.x for parallel computation
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index is within bounds
    if (index < N) { 
        c[index] = a[index] + b[index];
    }
}
```
