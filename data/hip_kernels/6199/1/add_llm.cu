#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c){
    // Calculate unique thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Perform addition for the calculated index
    c[idx] = a[idx] + b[idx];
}