#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define W 4000
#define H 20530

__global__ void calcmean(float *matrix, float *mean) {
    // Calculate row index using blockIdx.y and threadIdx.y
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate column index using blockIdx.x and threadIdx.x
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the kernel has a valid row and column index based on the size
    if (row < H && col < W) {
        // Atomic operation to increment the mean for the current row
        atomicAdd(&mean[row], matrix[row * W + col] / W);
    }
}