#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void floatToDouble(float* input, double* output, int numElements)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop to reduce instruction overhead
    #pragma unroll
    for (; i < numElements; i += blockDim.x * gridDim.x)
    {
        output[i] = static_cast<double>(input[i]);
    }
}