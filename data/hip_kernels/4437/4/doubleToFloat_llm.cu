#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void doubleToFloat(double* input, float* output, int numElements)
{
    // Calculate the global thread index
    int i = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;

    // Use a stride loop to allow each thread to process multiple elements
    int stride = hipGridDim_x * hipBlockDim_x;
    for (; i < numElements; i += stride) 
    {
        output[i] = static_cast<float>(input[i]);
    }
}