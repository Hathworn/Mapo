#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(float* d_in, float* d_out) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Use blockIdx and blockDim for global indexing
    if (idx < gridDim.x * blockDim.x) {              // Ensure idx is within bounds
        float f = d_in[idx];
        d_out[idx] = f * f;
    }
}