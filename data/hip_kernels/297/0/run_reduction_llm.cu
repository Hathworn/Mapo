#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void run_reduction(int *con, int *blockCon, int* ActiveList, int nActiveBlock, int* blockSizes)
{
    int list_idx = blockIdx.y * gridDim.x + blockIdx.x;

    if (list_idx < nActiveBlock)
    {
        int block_idx = ActiveList[list_idx];

        __shared__ int s_conv[REDUCTIONSHARESIZE];

        uint base_addr = block_idx * blockDim.x * 2; // *2 because there are only half block size number of threads
        uint tx = threadIdx.x;

        // Load two elements per thread into shared memory
        s_conv[tx] = con[base_addr + tx];
        s_conv[tx + blockDim.x] = con[base_addr + tx + blockDim.x];

        __syncthreads();

        // Perform reduction in shared memory
        for (uint i = blockDim.x; i > 0; i /= 2)
        {
            // Use warp shuffle to reduce synchronization overhead within a warp
            if (tx < i)
            {
                s_conv[tx] = (s_conv[tx] && s_conv[tx + i]) ? 1 : 0;
            }
            __syncthreads();
        }

        // Write the result of reduction to blockCon
        if (tx == 0)
        {
            blockCon[block_idx] = s_conv[0]; // active list is negation of tile convergence (active = not converged)
        }
    }
}