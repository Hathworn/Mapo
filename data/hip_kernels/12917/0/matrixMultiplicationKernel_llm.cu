#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void matrixMultiplicationKernel(long* A, long* B, long* C, long N) {
    long ROW = (blockIdx.y * blockDim.y) + threadIdx.y;
    long COL = (blockIdx.x * blockDim.x) + threadIdx.x;

    long tmpSum = 0;

    // Check bounds to ensure we do not access out of matrix bounds
    if (ROW < N && COL < N) {
        // Optimize by unrolling the loop when N is known and divisible
        for (long i = 0; i < N; i++) {
            tmpSum += A[ROW * N + i] * B[i * N + COL];
        }
        C[ROW * N + COL] = tmpSum;
    }
}