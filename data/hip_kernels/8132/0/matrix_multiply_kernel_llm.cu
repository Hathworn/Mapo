#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//Cuda checks
__global__ void matrix_multiply_kernel(unsigned char *temp, unsigned char *matrix, float *kernal, int order, int middle, int windowSizeX, int windowSizeY) {
    // Find place in the execution
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y >= windowSizeY || x >= windowSizeX) {
        return;
    }

    float sum = 0.0f;

    // Loop unrolling for better performance
    int orderSquared = order * order;
    int tempOffset, kernelOffset;
    for (int idx = 0; idx < orderSquared; ++idx) {
        int y2 = idx / order;
        int x2 = idx % order;
        
        int tempX = max(0, min(windowSizeX - 1, x - middle + x2)); // Apply boundary conditions
        int tempY = max(0, min(windowSizeY - 1, y - middle + y2)); // Apply boundary conditions

        tempOffset = (windowSizeX * tempY) + tempX;
        kernelOffset = (order * x2) + y2;
        sum += temp[tempOffset] * kernal[kernelOffset];
    }

    // Clamp the sum value using fminf and fmaxf
    sum = fminf(fmaxf(sum, 0.0f), 255.0f);

    // Add sum value to matrix
    matrix[(windowSizeX * y) + x] = (unsigned char) sum;
}