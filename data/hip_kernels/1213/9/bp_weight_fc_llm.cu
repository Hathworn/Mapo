#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bp_weight_fc(float *d_weight, float *d_preact, float *p_output, const int size, const int in_channel, const int out_channel)
{
    // Calculate global thread ID
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate total number of threads
    const int totalThreads = blockDim.x * gridDim.x;

    // Pre-calculate channel size for weights
    const int weight_channel = out_channel * in_channel;
    
    // Calculate total number of elements
    const int N = out_channel * in_channel * size * size;

    // Iterate over workload distributed among threads
    for (int n = N * tid / totalThreads; n < N * (tid + 1) / totalThreads; ++n) {
        int idx = n;
        // Compute indices for accessing weights and inputs
        const int i_channel = ((idx /= 1) % weight_channel);
        const int i_row = ((idx /= weight_channel) % size);
        const int i_col = ((idx /= size) % size);

        // Access and update weight
        d_weight[(i_channel * size + i_col) * size + i_row] = 
            d_preact[i_channel % out_channel] * 
            p_output[((i_channel % in_channel) * size + i_col) * size + i_row];
    }
}