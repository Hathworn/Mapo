#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fp_bias_fc(float *preact, float *bias, const int n_channel)
{
    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check within boundaries before operation
    if (idx < n_channel) {
        preact[idx] += bias[idx]; // Perform bias addition
    }
}