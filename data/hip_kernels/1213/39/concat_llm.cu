#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void concat(float* input1, float* input2, float* input3, float* input4, size_t num1, size_t num2, size_t num3, size_t num4, size_t maxNum, float* output, const int numPerBatch)
{
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x; // Calculate stride for iteration

    while (i < maxNum) {
        size_t batchIdx = i / numPerBatch; // Determine batch index
        const int batchOffset = i % numPerBatch; // Offset within the batch

        // Utilize conditional execution to improve performance
        if (batchOffset < num1) {
            output[i] = input1[batchOffset + batchIdx * num1];
        }
        else if (batchOffset < (num1 + num2)) {
            output[i] = input2[(batchOffset - num1) + batchIdx * num2];
        }
        else if (batchOffset < (num1 + num2 + num3)) {
            output[i] = input3[(batchOffset - (num1 + num2)) + batchIdx * num3];
        }
        else {
            output[i] = input4[(batchOffset - (num1 + num2 + num3)) + batchIdx * num4];
        }

        i += stride; // Move to the next iteration position using stride
    }
}