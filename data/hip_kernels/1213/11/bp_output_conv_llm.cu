#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bp_output_conv(float *d_output, float *weight, float *nd_preact, const int size, const int kernel_size, const int n_size, const int in_channel, const int out_channel, bool CONV, bool SAME)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalPos = blockDim.x * gridDim.x;
    const int N = kernel_size * kernel_size * size * size * in_channel * out_channel;
    const int weight_channel = out_channel * in_channel;
    const int padding = (kernel_size - 1) / 2;   // Must be int

    // Unroll loops to reduce control overhead
    for (int n = N * pos / totalPos; n < N * (pos + 1) / totalPos; ++n) {
        int idx = n;
        const int i_channel = (idx % weight_channel); idx /= weight_channel;
        const int i_kernel_row = (idx % kernel_size); idx /= kernel_size;
        const int i_kernel_col = (idx % kernel_size); idx /= kernel_size;
        const int i_row = (idx % size); idx /= size;
        const int i_col = idx % size;

        // Reduce redundant calculations by pre-computing indices
        int bpinput_row = i_kernel_row + i_row - (SAME ? padding : 2 * padding);
        int bpinput_col = i_kernel_col + i_col - (SAME ? padding : 2 * padding);

        if (bpinput_row >= 0 && bpinput_row < n_size && bpinput_col >= 0 && bpinput_col < n_size) {
            atomicAdd(&d_output[((i_channel % in_channel) * size + i_col) * size + i_row],
                      weight[(i_channel * kernel_size + (kernel_size - 1 - i_kernel_col)) * kernel_size + kernel_size - 1 - i_kernel_row]
                      * nd_preact[((i_channel % out_channel) * n_size + bpinput_col) * n_size + bpinput_row]);
        }
    }
}