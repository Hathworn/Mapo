#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bp_weight_conv(float* d_weight, float* d_preact, float* p_output, const int kernel_size, const int size, const int n_size, const int in_channel, const int out_channel, bool SAME)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalPos = blockDim.x * gridDim.x;
    const int N = kernel_size * kernel_size * n_size * n_size * in_channel * out_channel;  // total number of connections in this convolution
    const int weight_channel = in_channel * out_channel;
    const int padding = (kernel_size - 1) / 2;
    int input_row, input_col;

    // calculate thread range
    int n_start = N * pos / totalPos;
    int n_end = N * (pos + 1) / totalPos;

    // loop through assigned connections
    for (int n = n_start; n < n_end; n++) {
        int idx = n;
        const int i_kernel_row = idx % kernel_size;
        idx /= kernel_size;
        const int i_kernel_col = idx % kernel_size;
        idx /= kernel_size;
        const int i_channel = idx % weight_channel;
        idx /= weight_channel;
        const int i_row = idx % n_size;
        const int i_col = idx / n_size;

        // calculate input index
        if (SAME) {
            input_row = i_kernel_row + i_row - padding;
            input_col = i_kernel_col + i_col - padding;
        } else {
            input_row = i_kernel_row + i_row;
            input_col = i_kernel_col + i_col;
        }

        if (input_row >= 0 && input_row < size && input_col >= 0 && input_col < size) {
            atomicAdd(&d_weight[(i_channel * kernel_size + i_kernel_col) * kernel_size + i_kernel_row],
                      d_preact[((i_channel % out_channel) * n_size + i_col) * n_size + i_row] * p_output[((i_channel % in_channel) * size + input_col) + input_row]);
        }
    }
}