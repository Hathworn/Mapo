#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fp_bias_conv(float* preact, float* bias, const int size, const int n_channel)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalPos = blockDim.x * gridDim.x;

    const int N = n_channel * size * size;

    // Calculate workload for each thread
    int start = (N * pos) / totalPos;
    int end = (N * (pos + 1)) / totalPos;

    // Use appropriate stride access for coalesced memory access
    for (int n = start; n < end; ++n) {
        int idx = n;
        const int i_channel = (idx / (size * size)) % n_channel;
        const int i_row = (idx / size) % size;
        const int i_col = idx % size;

        // Precompute linear index for memory access
        int preact_idx = (i_channel * size * size) + (i_row * size) + i_col;
        preact[preact_idx] += bias[i_channel];
    }
}