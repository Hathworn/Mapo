#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumGrad(float* output, float* input1, float* input2, float* input3, float* input4, const int numElem)
{
    // Calculate global thread ID
    size_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    // Use stride to allow threads to traverse the dataset in case of underutilization
    size_t stride = blockDim.x * gridDim.x;

    for(size_t i = tid; i < numElem; i += stride) {
        // Perform the addition operation
        output[i] = input1[i] + input2[i] + input3[i] + input4[i];
    }
}