#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fp_preact_fc(float* input, float* preact, float* weight, const int size, const int in_channel, const int out_channel)
{
    // Calculate global thread index
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalPos = blockDim.x * gridDim.x;

    // Weight matrix size calculations
    const int weight_channel = in_channel * out_channel;
    const int N = out_channel * in_channel * size * size; 

    // Calculate starting and ending indices for threads
    int startIndex = (N * pos) / totalPos;
    int endIndex = (N * (pos + 1)) / totalPos;

    // Loop over assigned elements
    for (int n = startIndex; n < endIndex; ++n) {
        int idx = n;

        // Calculate indices for accessing arrays
        const int i_channel = (idx % weight_channel);
        idx /= weight_channel;
        const int i_row = (idx % size);
        idx /= size;
        const int i_col = (idx % size);

        // Perform atomic addition to preact array
        atomicAdd(&preact[i_channel % out_channel], 
                  weight[(i_channel * size + i_col) * size + i_row] * 
                  input[((i_channel % in_channel) * size + i_col) * size + i_row]);
    }
}