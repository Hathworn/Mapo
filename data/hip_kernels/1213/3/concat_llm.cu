#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void concat(float* output, float* input1, float* input2, float* input3, float* input4, const int size, const int in_channel1, const int in_channel2, const int in_channel3, const int in_channel4)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int out_channel = in_channel1 + in_channel2 + in_channel3 + in_channel4;  // # of channel for output
    const int N = size * size;  // total elements per channel

    // Use dynamic boundary check to optimize performance
    if (pos < N * out_channel) {  
        const int n = pos / N; // Determine channel index
        const int idx = pos % N; // Determine position in channel
        const int row = idx / size;
        const int col = idx % size;

        // Efficiently map input based on dynamic channel index
        if (n < in_channel1) {  
            output[pos] = input1[pos];
        } else if (n < in_channel1 + in_channel2) {  
            output[pos] = input2[pos - in_channel1 * N];
        } else if (n < in_channel1 + in_channel2 + in_channel3) {  
            output[pos] = input3[pos - (in_channel1 + in_channel2) * N];
        } else {  
            output[pos] = input4[pos - (in_channel1 + in_channel2 + in_channel3) * N];
        }
    }
}