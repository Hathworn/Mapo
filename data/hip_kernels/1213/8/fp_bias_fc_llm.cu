#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fp_bias_fc(float *preact, float *bias, const int n_channel)
{
    // Calculate the global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index does not exceed the number of channels
    if (idx < n_channel) {
        // Add bias to the corresponding preact element
        preact[idx] += bias[idx];
    }
}