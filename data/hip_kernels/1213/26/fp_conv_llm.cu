#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fp_conv(float* output, float* input, float* weight, const int kernel_size, const int size, const int n_size, const int in_channel, const int out_channel, bool SAME)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalPos = blockDim.x * gridDim.x;
    const int N = kernel_size * kernel_size * n_size * n_size * in_channel * out_channel;
    const int weight_channel = in_channel * out_channel;  
    const int padding = (kernel_size - 1) / 2;  

    // Optimize: Loop stride step greater than 1 for better utilization
    for(int n = N * pos / totalPos; n < N * (pos+1) / totalPos; n += blockDim.x)
    {
        int idx = n;
        const int i_kernel_row = (idx % kernel_size); idx /= kernel_size;
        const int i_kernel_col = (idx % kernel_size); idx /= kernel_size;
        const int i_channel = (idx % weight_channel); idx /= weight_channel;
        const int i_row = (idx % n_size); idx /= n_size;
        const int i_col = (idx % n_size);

        int input_row, input_col;

        if (SAME)
        {
            input_row = i_kernel_row + i_row - padding;
            input_col = i_kernel_col + i_col - padding;
        }
        else
        {
            input_row = i_kernel_row + i_row;
            input_col = i_kernel_col + i_col;
        }

        if(input_row >= 0 && input_row < size && input_col >=0 && input_col < size)
        {
            atomicAdd(&output[((i_channel % out_channel) * n_size + i_col) * n_size + i_row],
                      weight[(i_channel * kernel_size + i_kernel_col) * kernel_size + i_kernel_row]
                      * input[((i_channel % in_channel) * size + input_col) * size + input_row]);
        }
    }
}