#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fp_bias_conv(float* preact, float* bias, const int size, const int n_channel)
{
    // Calculate the global thread index
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalPos = blockDim.x * gridDim.x;
    const int N = n_channel * size * size;

    // Optimize loop by reducing index computations
    for (int n = N * pos / totalPos; n < N * (pos + 1) / totalPos; ++n) {
        int idx = n;
        int i_col = idx % size; // Combined row and column calculation
        idx /= size;
        int i_row = idx % size;
        idx /= size;
        int i_channel = idx % n_channel;

        // Linear index calculation for memory access
        preact[(i_channel * size + i_col) * size + i_row] += bias[i_channel];
    }
}