#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcLoss(float *err, float *output, unsigned int Y, const int N)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    // Iterate over the entire range using thread stride to maximize utilization
    for (int idx = pos; idx < N; idx += stride) {
        err[idx] = ((Y == idx ? 1.0f : 0.0f) - output[idx]);
    }
}