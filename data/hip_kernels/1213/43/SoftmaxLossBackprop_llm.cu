#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SoftmaxLossBackprop(const float *label, int num_labels, int batch_size, float *diff)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= batch_size) 
        return;

    const int label_value = static_cast<int>(label[idx]);

    // Utilize shared memory to coalesce global memory accesses
    extern __shared__ float shared_diff[];
    shared_diff[threadIdx.x] = diff[idx * num_labels + label_value];
    __syncthreads();

    // Decrease the result of the label's value by 1
    shared_diff[threadIdx.x] -= 1.0f;
    diff[idx * num_labels + label_value] = shared_diff[threadIdx.x];
}