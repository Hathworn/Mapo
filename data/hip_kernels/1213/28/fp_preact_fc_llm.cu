#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fp_preact_fc(float* input, float* preact, float* weight, const int size, const int in_channel, const int out_channel)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int weight_channel = in_channel * out_channel;
    const int N = out_channel * in_channel * size * size;  // number of elements of weight matrix

    for (int n = pos; n < N; n += blockDim.x * gridDim.x) {  // Use a stride loop
        int idx = n;
        const int i_channel = idx % weight_channel;
        idx /= weight_channel;
        const int i_row = idx % size;
        idx /= size;
        const int i_col = idx % size;

        // Pre-compute indices and eliminate redundant calculations
        int weight_idx = (i_channel * size + i_col) * size + i_row;
        int input_idx = ((i_channel % in_channel) * size + i_col) * size + i_row;

        atomicAdd(&preact[i_channel % out_channel], weight[weight_idx] * input[input_idx]);
    }
}