#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bp_weight_fc(float *d_weight, float *d_preact, float *p_output, const int size, const int in_channel, const int out_channel)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalPos = blockDim.x * gridDim.x;
    const int N = out_channel * in_channel * size * size;
    const int weight_channel = out_channel * in_channel;

    for (int n = N * pos / totalPos; n < N * (pos + 1) / totalPos; ++n) {
        int idx = n;
        const int i_channel = idx % weight_channel; // Simplified {idx/=1} to {idx%weight_channel}
        idx /= weight_channel;
        const int i_row = idx % size;
        idx /= size;
        const int i_col = idx % size;

        // Precomputed indices to reduce repeated calculation
        const int preact_idx = i_channel % out_channel;
        const int output_idx = ((i_channel % in_channel) * size + i_col) * size + i_row;

        // Removed redundant calculations in the assignment expression
        d_weight[(i_channel * size + i_col) * size + i_row] = d_preact[preact_idx] * p_output[output_idx];
    }
}