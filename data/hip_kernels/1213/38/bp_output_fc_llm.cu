#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bp_output_fc(float *d_output, float *d_preact, float *weight, const int size, const int in_channel, const int out_channel)
{
    // Calculate unique thread index using both block and thread indices
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    const int weight_channel = out_channel * in_channel;
    const int N = weight_channel * size * size;

    // Loop over the assigned range and increment by the total number of threads
    for (int n = idx; n < N; n += blockDim.x * gridDim.x) {
        int index = n;
        // Calculate indices for i_channel, i_row, and i_col
        const int i_channel = index / (size * size);
        const int i_row = (index / size) % size;
        const int i_col = index % size;

        // Use atomic addition to prevent race conditions
        atomicAdd(&d_output[((i_channel % in_channel) * size + i_col) * size + i_row], 
                  d_preact[i_channel % out_channel] * weight[n]);
    }
}