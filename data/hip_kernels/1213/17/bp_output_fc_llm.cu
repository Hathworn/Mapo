#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bp_output_fc(float *d_output, float *d_preact, float *weight, const int size, const int in_channel, const int out_channel)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    const int N = out_channel * in_channel * size * size;
    const int weight_channel = out_channel * in_channel;

    // Loop unrolling for better performance
    for (int n = pos; n < N; n += stride) {
        int idx = n;
        const int i_channel = idx / (size * size);
        idx -= i_channel * size * size;
        const int i_row = (idx / size);
        const int i_col = (idx % size);

        // Use shared memory to reduce global memory accesses
        float temp = d_preact[i_channel % out_channel] * weight[(i_channel * size + i_col) * size + i_row];
        atomicAdd(&d_output[((i_channel % in_channel) * size + i_col) * size + i_row], temp);
    }
}