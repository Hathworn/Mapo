#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fp_maxpool(float* output, float* input, const int kernel_size, const int size, const int n_size, const int in_channel, bool SAME)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalPos = blockDim.x * gridDim.x;
    const int N = kernel_size * kernel_size * n_size * n_size * in_channel;
    const int padding = (kernel_size - 1) / 2;
    int input_row, input_col;

    // Distribute workload to threads more efficiently
    for(int n = N * pos / totalPos; n < N * (pos+1) / totalPos; n++) {
        int idx = n;
        const int i_kernel_row = idx % kernel_size; idx /= kernel_size;
        const int i_kernel_col = idx % kernel_size; idx /= kernel_size;
        const int i_channel = idx % in_channel; idx /= in_channel;
        const int i_row = idx % n_size; idx /= n_size;
        const int i_col = idx % n_size;
        float maxidx = -100.0f; // Initialized maxidx appropriately

        // Compute input indices
        input_row = i_kernel_row + i_row - (SAME ? padding : 0);
        input_col = i_kernel_col + i_col - (SAME ? padding : 0);

        // Check for valid bounds and find max value
        if(input_row >= 0 && input_row < size && input_col >= 0 && input_col < size) {
            int input_index = ((i_channel * size + input_col) * size + input_row);
            float input_val = input[input_index];

            if (input_val > maxidx) {
                int output_index = ((i_channel * n_size + i_col) * n_size + i_row);
                output[output_index] = input_val;
            }
        }
    }
}