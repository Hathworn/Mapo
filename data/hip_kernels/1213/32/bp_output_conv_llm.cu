#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bp_output_conv(float *d_output, float *weight, float *nd_preact, const int size, const int kernel_size, const int n_size, const int in_channel, const int out_channel, bool CONV, bool SAME)
{
    // Calculate global thread position
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalPos = blockDim.x * gridDim.x;

    // Pre-calculate constants
    const int N = kernel_size * kernel_size * size * size * in_channel * out_channel;
    const int weight_channel = out_channel * in_channel;
    const int padding = (kernel_size - 1) / 2;

    for (int n = N * pos / totalPos; n < N * (pos + 1) / totalPos; ++n) {
        int idx = n;

        // Sequential access optimization
        const int i_channel = idx % weight_channel;
        idx /= weight_channel;
        const int i_kernel_row = idx % kernel_size;
        idx /= kernel_size;
        const int i_kernel_col = idx % kernel_size;
        idx /= kernel_size;
        const int i_row = idx % size;
        idx /= size;
        const int i_col = idx % size;

        // Determine input position based on padding scheme
        const int bpinput_row = i_kernel_row + i_row - (SAME ? padding : 2 * padding);
        const int bpinput_col = i_kernel_col + i_col - (SAME ? padding : 2 * padding);

        // Update output if within bounds
        if (bpinput_row >= 0 && bpinput_row < n_size && bpinput_col >= 0 && bpinput_col < n_size) {
            atomicAdd(&d_output[((i_channel % in_channel) * size + i_col) * size + i_row],
                      weight[(i_channel * kernel_size + (kernel_size - 1 - i_kernel_col)) * kernel_size + kernel_size - 1 - i_kernel_row]
                      * nd_preact[((i_channel % out_channel) * n_size + bpinput_col) * n_size + bpinput_row]);
        }
    }
}