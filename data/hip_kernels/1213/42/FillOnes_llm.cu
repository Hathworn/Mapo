#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FillOnes(float *vec, int size)
{
    // Calculate unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (idx < size) {
        vec[idx] = 1.0f; // Initialize to 1.0f
    }
}