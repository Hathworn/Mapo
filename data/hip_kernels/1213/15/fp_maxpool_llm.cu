#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fp_maxpool(float* output, float* input, const int kernel_size, const int size, const int n_size, const int in_channel, bool SAME)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalPos = blockDim.x * gridDim.x;
    const int N = kernel_size * kernel_size * n_size * n_size * in_channel;  // total number of connections in this convolution
    const int padding = (kernel_size - 1) / 2;  // number of padding for both ends
    
    for(int n = N * pos / totalPos; n < N * (pos+1) / totalPos; n++) {
        int idx = n;
        const int i_kernel_row = idx % kernel_size;
        idx /= kernel_size;
        const int i_kernel_col = idx % kernel_size;
        idx /= kernel_size;
        const int i_channel = idx % in_channel;
        idx /= in_channel;
        const int i_row = idx % n_size;
        const int i_col = idx / n_size;  // avoids recomputation
        float maxidx = (float)-100;

        // Calculate input matrix positions
        int input_row = i_kernel_row + i_row - (SAME ? padding : 0);
        int input_col = i_kernel_col + i_col - (SAME ? padding : 0);
        
        if(input_row >= 0 && input_row < size && input_col >= 0 && input_col < size) {
            int input_index = ((i_channel * size + input_col) * size + input_row);
            if (input[input_index] > maxidx) {
                output[((i_channel * n_size + i_col) * n_size + i_row)] = input[input_index];
            }
        }
    }
}