#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumGrad(float* input1, float* input2, float* input3, float* input4, float* output, const int numElem)
{
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimize by removing the size calculation and simplify loop
    if (i < numElem) {
        output[i] = input1[i] + input2[i] + input3[i] + input4[i];
    }
}