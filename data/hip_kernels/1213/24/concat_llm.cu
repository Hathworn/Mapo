#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void concat(float* output, float* input1, float* input2, float* input3, float* input4, const int size, const int in_channel1, const int in_channel2, const int in_channel3, const int in_channel4)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int out_channel = in_channel1 + in_channel2 + in_channel3 + in_channel4;  // # of channel for output
    const int N = size * size;  // total elements per channel

    if (pos < N) {
        const int row = pos / size;
        const int col = pos % size;
        
        // Use the stride to reduce multiple calculations
        const int offset = col * size + row;

        for (int n = 0; n < out_channel; n++) {
            if (n < in_channel1) {  // first input
                output[n * N + offset] = input1[n * N + offset];
            } 
            else if (n < in_channel1 + in_channel2) {  // second input
                output[n * N + offset] = input2[(n - in_channel1) * N + offset];
            } 
            else if (n < in_channel1 + in_channel2 + in_channel3) {  // third input
                output[n * N + offset] = input3[(n - in_channel1 - in_channel2) * N + offset];
            } 
            else {  // last input
                output[n * N + offset] = input4[(n - in_channel1 - in_channel2 - in_channel3) * N + offset];
            }
        }
    }
}