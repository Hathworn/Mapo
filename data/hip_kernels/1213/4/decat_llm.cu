#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void decat(float* input, float* output1, float* output2, float* output3, float* output4, const int size, const int out_channel1, const int out_channel2, const int out_channel3, const int out_channel4)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int in_channel = out_channel1 + out_channel2 + out_channel3 + out_channel4;  // total channels of input
    const int N = size * size;  // total elements per channel

    if (pos < N) {
        // Calculate row and col only once
        const int row = pos / size;
        const int col = pos % size;

        for (int n = 0; n < in_channel; n++) {
            // Pre-calculate index for input and avoid repeated calculation
            const int inputIndex = (n * size + col) * size + row;
            
            // Output to the correct array based on channel range
            if (n < out_channel1) {
                output1[inputIndex] = input[inputIndex];  // first output
            } else if (n < out_channel1 + out_channel2) {
                output2[(n - out_channel1) * N + pos] = input[inputIndex];  // second output
            } else if (n < out_channel1 + out_channel2 + out_channel3) {
                output3[(n - out_channel1 - out_channel2) * N + pos] = input[inputIndex];  // third output
            } else {
                output4[(n - out_channel1 - out_channel2 - out_channel3) * N + pos] = input[inputIndex];  // last output
            }
        }
    }
}