#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bp_maxpool(float* d_preact, float* preact, float* p_output, float* nd_output, const int kernel_size, const int size, const int n_size, const int in_channel, bool SAME)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalPos = blockDim.x * gridDim.x;
    const int N = kernel_size * kernel_size * n_size * n_size * in_channel;  // total number of connections in this convolution
    const int padding = (kernel_size - 1) / 2;  // number of padding for both ends

    // distribute certain number of connections to each thread regardless of detailed position and shape
    for (int n = N * pos / totalPos; n < N * (pos + 1) / totalPos; n++)
    {
        int idx = n;
        const int i_kernel_row = (idx % kernel_size);
        const int i_kernel_col = ((idx / kernel_size) % kernel_size);
        const int i_channel = ((idx / (kernel_size * kernel_size)) % in_channel);
        const int i_row = ((idx / (kernel_size * kernel_size * in_channel)) % n_size);
        const int i_col = ((idx / (kernel_size * kernel_size * in_channel * n_size)) % n_size);

        float maxidx = -1.0f;  // Use float literal
        int maxpos = 0;        // Track max position

        // corresponding position of the input matrix
        int input_row, input_col;
        if (SAME) // SAME padding scheme implemented
        {
            input_row = i_kernel_row + i_row - padding;
            input_col = i_kernel_col + i_col - padding;
        }
        else
        {
            input_row = i_kernel_row + i_row;
            input_col = i_kernel_col + i_col;
        }

        if (input_row >= 0 && input_row < size && input_col >= 0 && input_col < size)
        {
            // Simplify indexing calculation and reduce redundant modulo operation
            int current_idx = ((i_channel * size + input_col) * size + input_row);
            if (p_output[current_idx] > maxidx)
            {
                maxidx = p_output[current_idx];
                maxpos = current_idx;
            }
        }
        d_preact[maxpos] = nd_output[((i_channel * n_size + i_col) * n_size + i_row)];
    }
}