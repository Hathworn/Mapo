#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void decat(float* output1, float* output2, float* output3, float* output4, size_t num1, size_t num2, size_t num3, size_t num4, size_t maxNum, float* input, const int numPerBatch)
{
    size_t i = blockDim.x * blockIdx.x + threadIdx.x;

    while(i < maxNum) {
        // Calculate batchIdx and batchOffset
        size_t batchIdx = i / numPerBatch;
        const int batchOffset = i % numPerBatch;

        // Use a tmp variable to reduce repetitive calculations
        if(batchOffset < num1) {
            output1[batchOffset + batchIdx * num1] = input[i];
        } 
        else if(batchOffset < num1 + num2) {
            output2[batchOffset - num1 + batchIdx * num2] = input[i];
        } 
        else if(batchOffset < num1 + num2 + num3) {
            output3[batchOffset - num1 - num2 + batchIdx * num3] = input[i];
        } 
        else {
            output4[batchOffset - num1 - num2 - num3 + batchIdx * num4] = input[i];
        }

        i += blockDim.x * gridDim.x; // Update i for next loop iteration
    }
}