#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcLoss(float *err, float *output, unsigned int Y, const int N)
{
    // Calculate the global thread index.
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within the bounds.
    if (idx < N) {
        // Compute the error directly.
        err[idx] = ((Y == idx ? 1.0f : 0.0f) - output[idx]);
    }
}