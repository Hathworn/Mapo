#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bp_maxpool(float* d_preact, float* preact, float* p_output, float* nd_output, const int kernel_size, const int size, const int n_size, const int in_channel, bool SAME)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalPos = blockDim.x * gridDim.x;
    const int N = kernel_size * kernel_size * n_size * n_size * in_channel;
    const int padding = (kernel_size - 1) / 2;
    int input_row, input_col;

    // Use fewer divisions by storing initial division results
    const int n_start = N * pos / totalPos;
    const int n_end = N * (pos + 1) / totalPos;

    for(int n = n_start; n < n_end; n++){
        int idx = n;
        
        // Calculations with minimized operations.
        const int i_kernel_row = idx % kernel_size; idx /= kernel_size;
        const int i_kernel_col = idx % kernel_size; idx /= kernel_size;
        const int i_channel   = idx % in_channel; idx /= in_channel;
        const int i_row       = idx % n_size; idx /= n_size;
        const int i_col       = idx % n_size;

        float maxidx = (float)-1;
        idx = 0;

        // Recompute input positions based on padding
        if (SAME){
            input_row = i_kernel_row + i_row - padding;
            input_col = i_kernel_col + i_col - padding;
        } else {
            input_row = i_kernel_row + i_row;
            input_col = i_kernel_col + i_col;
        }

        // Check bounds after computing input positions
        if(input_row >= 0 && input_row < size && input_col >=0 && input_col < size){
            int output_idx = ((i_channel % in_channel) * size + input_col) * size + input_row;
            if (p_output[output_idx] > maxidx) {
                maxidx = p_output[output_idx];
                idx = output_idx;
            }
        }

        d_preact[idx] = nd_output[((i_channel % in_channel) * n_size + i_col) * n_size + i_row];
    }
}