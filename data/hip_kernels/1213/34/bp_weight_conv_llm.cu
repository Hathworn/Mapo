#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bp_weight_conv(float* d_weight, float* d_preact, float* p_output, const int kernel_size, const int size, const int n_size, const int in_channel, const int out_channel, bool SAME)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalPos = blockDim.x * gridDim.x;
    const int N = kernel_size * kernel_size * n_size * n_size * in_channel * out_channel;  // total number of connections in this convolution
    const int weight_channel = in_channel * out_channel;  // actual number of channels of weight matrix
    const int padding = (kernel_size - 1) / 2;  // number of padding for both ends

    // distribute certain number of connections to each thread regardless of detailed position and shape
    for(int n = N * pos / totalPos; n < N * (pos+1) / totalPos; n++){
        int idx = n;
        const int i_kernel_row = idx % kernel_size;
        idx /= kernel_size;
        const int i_kernel_col = idx % kernel_size;
        idx /= kernel_size;
        const int i_channel = idx % weight_channel;
        idx /= weight_channel;
        const int i_row = idx % n_size;
        idx /= n_size;
        const int i_col = idx % n_size;

        // corresponding position of the input matrix
        int input_row = i_kernel_row + i_row - (SAME ? padding : 0);  // Simplified row calculation
        int input_col = i_kernel_col + i_col - (SAME ? padding : 0);  // Simplified col calculation
        
        // Ensure input position is within bounds
        if(input_row >= 0 && input_row < size && input_col >=0 && input_col < size){
            atomicAdd(&d_weight[(i_channel * kernel_size + i_kernel_col) * kernel_size + i_kernel_row],
                      d_preact[((i_channel % out_channel) * n_size + i_col) * n_size + i_row] * p_output[((i_channel % in_channel) * size + input_col) + input_row]);
        }
    }
}