#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod, const size_t lod_len, size_t* out_lod0) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check index within bounds
    if (index < lod_len) {
        // Avoid redundant memory accesses by reading in_lod[index] only once
        size_t lod_val = in_lod[index];
        out_lod0[index] = lod_val - num_erased[lod_val];
    }
}