#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumaGPU(int a, int b, int *sol) {
    // Use shared memory to reduce global memory access latency
    __shared__ int tempResult;
    int threadId = threadIdx.x;

    if (threadId == 0) {
        tempResult = a + b;
    }
    __syncthreads();

    if (threadId == 0) {
        *sol = tempResult;
    }
}