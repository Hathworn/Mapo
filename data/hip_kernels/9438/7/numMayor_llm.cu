#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Global kernel function optimized
__global__ void numMayor(float *d_v, float *d_pos){

    // Use shared memory to reduce global memory accesses
    __shared__ float s_temp;
    __shared__ float s_pos;

    // Initialize shared memory
    if (threadIdx.x == 0) {
        s_temp = 0;
        s_pos = 0;
    }
    __syncthreads();

    // Use block-level threading with stride for better performance
    float local_temp = 0;
    float local_pos = 0;
    for (int i = threadIdx.x; i < blockDim.x; i += blockDim.x) {
        if (d_v[i] > local_temp) {
            local_temp = d_v[i];
            local_pos = i;
        }
    }

    // Cooperative reduction within the block
    atomicMax(&s_temp, local_temp);
    __syncthreads();
    if (s_temp == local_temp) {
        atomicMax(&s_pos, local_pos);
    }
    __syncthreads();

    // Write results to global memory if thread 0
    if (threadIdx.x == 0) {
        if (s_pos > *d_pos)
            *d_pos = s_pos;
        *d_v = s_temp;
    }
}