#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumaVectores (float * d_a, float *d_b, float * d_c, int n) {
    // Calculate the global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Check bounds to avoid extra threads computation
    if (index < n) {
        d_c[index] = d_a[index] + d_b[index];
    }
}
```
