#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void testKernel(float *g_idata, float *g_odata)
{
    // shared memory with size determined by the host application
    extern __shared__ float sdata[];

    // access thread id
    const unsigned int tid = threadIdx.x;
    // optimized using block and grid sizes for larger datasets
    const unsigned int idx = blockIdx.x * blockDim.x + tid;
    
    // read in input data from global memory
    sdata[tid] = g_idata[idx];
    __syncthreads();

    // perform computations
    sdata[tid] *= blockDim.x; // use *= for in-place multiply
    __syncthreads();

    // write data to global memory
    g_odata[idx] = sdata[tid];
}