#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 24

__global__ void muestraIndice(float *a, float *b, float *c){

    int global = blockIdx.x * blockDim.x + threadIdx.x;

    // Utilize shared memory for computations to potentially reduce fetch latency
    __shared__ float shared_a[blockDim.x];
    __shared__ float shared_b[blockDim.x];

    if(global < N){
        shared_a[threadIdx.x] = threadIdx.x;
        shared_b[threadIdx.x] = blockIdx.x;
        c[global] = global;

        // Copy computed data from shared memory to global memory
        a[global] = shared_a[threadIdx.x];
        b[global] = shared_b[threadIdx.x];
    }
}