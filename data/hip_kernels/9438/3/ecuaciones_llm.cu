#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ecuaciones(int a, int b, int c, float *sol) {
    // Utilize blockIdx.x and blockDim.x for parallel processing
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    float d = 0;
    float x = 0, y = 0;

    // Precompute 2*a once for reuse
    float two_a = 2.0f * a;

    d = b * b - 4 * a * c;
    if (d > 0) {
        // Compute sqrt(d) once for reuse
        float sqrt_d = sqrt(d);
        x = (-b + sqrt_d) / two_a;
        y = (-b - sqrt_d) / two_a;
        sol[index * 2] = x; // Store results in consecutive memory locations
        sol[index * 2 + 1] = y;
    }
    else if (d == 0) {
        x = (-b) / two_a;
        sol[index * 2] = x; // Ensure consistent memory access pattern
    }
}