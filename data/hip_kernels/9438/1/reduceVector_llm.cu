#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceVector(float *v1, float *v2, float *res){

    // Get global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for faster access
    extern __shared__ float shared[];

    // Load elements into shared memory
    shared[threadIdx.x] = v1[index];
    __syncthreads();

    // Perform reduction using shared memory
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (threadIdx.x < i) {
            shared[threadIdx.x] += shared[threadIdx.x + i];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0)
        res[blockIdx.x] = shared[0];
}