#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumaVector(float *v1, float *v2, float *res) {
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll loop for performance improvement, assume TAM_V is divisible by block size
    #pragma unroll
    for (int i = index; i < TAM_V; i += blockDim.x * gridDim.x) {
        res[i] = v1[i] + v2[i];
    }
}