#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define K 3
#define BLCH 8
#define BLCW 32

__global__ void compute_gpu(float *img, float *f, float *out, int bh, int bw, int imgH, int imgW, int imgN, int nF, int convH, int convW){
    int idY = blockDim.y * blockIdx.y + threadIdx.y;
    int idX = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimize indexing by precomputing offsets
    int imgSize = imgW * imgH;
    int convSize = convW * convH;
    int filterSize = nF * nF;

    for (int mi = 0; mi < imgN; mi++) {
        int inm1 = mi * imgSize; // Fixed offset for current image
        int ind1 = mi * convSize; // Fixed offset for output

        if (idX < convH && idY < convW) {
            int ind3 = ind1 + idX * convW + idY; // Output index within the current feature map
            int inm3 = inm1 + idX * imgW + idY; // Image index within the current image

            for (int fi = 0; fi < nF; fi++) {
                int inm4 = inm3 + imgW * fi; // Image index offset by filter row
                int inf = (ind3 * nF + fi) * nF; // Filter index within the current filter

                // Perform convolution
                float sum = 0.0f; // Use a local accumulator to optimize memory access
                for (int fj = 0; fj < nF; fj++) {
                    sum += img[inm4 + fj] * f[inf + fj];
                }
                // Write accumulated result to prevent multiple accesses to global memory
                out[ind3] += sum;
            }
        }
    }
}