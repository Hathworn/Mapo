#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expandPlanes_kernel_Fp32_NCHW(float* output, const uint64_t* masks, const float* values, int n) {
    // Define constants for better clarity
    constexpr int kThreadsPerPlane = 64;
    constexpr int kNumShmemElements = 4; // 256 / 64

    __shared__ uint64_t shMasks[kNumShmemElements];
    __shared__ float shVals[kNumShmemElements];

    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int planeIndex = index >> 6;

    if (planeIndex >= n) return;

    // Load inputs to shared memory - each shmem element handles 64 threads
    if (threadIdx.x < kNumShmemElements) {
        shMasks[threadIdx.x] = masks[planeIndex + threadIdx.x];
        shVals[threadIdx.x] = values[planeIndex + threadIdx.x];
    }
    __syncthreads();

    // Load the mask and value for the current thread's segment
    int shmemIndex = threadIdx.x >> 6;
    uint64_t mask = shMasks[shmemIndex];
    float op = 0;

    // Determine if the output at this index should be set
    bool set = mask & (1ull << (index & 0x3F));
    if (set) {
        op = shVals[shmemIndex];
    }

    output[index] = op;
}