#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Early return for out-of-bounds threads
    if (i >= h || j >= w) return;

    const int pos = i * s + j;

    // Load normalization factor once
    float scale = normalization_factor[pos];

    // Avoid division if scale is zero
    float invScale = (scale == 0.0f) ? 1.0f : __frcp_rn(scale);

    // Update image value with precomputed inverse scale
    image[pos] *= invScale;
}