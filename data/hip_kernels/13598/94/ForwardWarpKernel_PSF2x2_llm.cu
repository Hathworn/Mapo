#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    // Precompute common indices
    int flow_row_offset  = i * flow_stride + j;
    int image_row_offset = i * image_stride + j;
    float value = src[image_row_offset];

    // Compute the position and offsets
    float cx = u[flow_row_offset] * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v[flow_row_offset] * time_scale + static_cast<float>(i) + 1.0f;

    // pixel containing bottom left corner
    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);

    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);

    // Macro to avoid repetitive code for checking bounds and updating
    #define UPDATE_PIXEL(cond, x_offset, y_offset, d_x, d_y) \
        if (cond) { \
            weight = (d_x) * (d_y); \
            int index = (ty + (y_offset)) * image_stride + (tx + (x_offset)); \
            _atomicAdd(dst + index, value * weight); \
            _atomicAdd(normalization_factor + index, weight); \
        }

    float weight;
    // Fill pixels with calculated weights
    UPDATE_PIXEL(!(tx >= w || tx < 0 || ty >= h || ty < 0), 0, 0, dx, dy)               // Bottom right
    UPDATE_PIXEL(!(tx - 1 >= w || tx - 1 < 0 || ty >= h || ty < 0), -1, 0, 1.0f - dx, dy) // Bottom left
    UPDATE_PIXEL(!(tx - 1 >= w || tx - 1 < 0 || ty - 1 >= h || ty - 1 < 0), -1, -1, 1.0f - dx, 1.0f - dy) // Upper left
    UPDATE_PIXEL(!(tx >= w || tx < 0 || ty - 1 >= h || ty - 1 < 0), 0, -1, dx, 1.0f - dy) // Upper right

    #undef UPDATE_PIXEL
}