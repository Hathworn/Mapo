#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate unique thread index
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if within bounds using a simple conditional
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];
        
        // Use register to store scaled values before writing to memory for efficiency
        float scaled_x = loc.x * scale;
        float scaled_y = loc.y * scale;

        x[ptidx] = scaled_x;
        y[ptidx] = scaled_y;
    }
}