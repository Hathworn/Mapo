#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate unique thread index in the grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Early exit if out of bounds
    if (idx >= w || idy >= h) return;

    // Compute linear memory location
    const int pos = idy * w + idx;

    // Assign the value
    image[pos] = value;
}