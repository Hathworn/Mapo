#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Check for a valid index to avoid unused threads
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N; // Only one thread performs the division
    }
}