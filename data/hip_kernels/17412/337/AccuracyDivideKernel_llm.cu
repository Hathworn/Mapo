#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory to reduce global memory access
    __shared__ float shared_accuracy;

    // Only one thread does the division
    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy / N;
    }

    // Synchronize to ensure shared memory is written before any thread proceeds
    __syncthreads();

    // All threads update the global memory with the result
    if (threadIdx.x == 0) {
        *accuracy = shared_accuracy;
    }
}