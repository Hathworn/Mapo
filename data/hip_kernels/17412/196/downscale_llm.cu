#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ __forceinline__ size_t idx(const size_t nc, const size_t height, const size_t width, const size_t y, const size_t x) {
    return (nc * height + y) * width + x;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x;
    ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
    if (ii >= no_elements) return;
    
    // Pre-compute scale factors for repeated use
    const int d2_scale = d2 * scale_factor;
    const int d3_scale = d3 * scale_factor;

    float gradInput_accum = 0.0f; // Accumulate in local variable
    for (int i = 0; i < scale_factor; i++) {
        for (int j = 0; j < scale_factor; j++) {
            int ipidx = (((ii / (d2 * d3)) * d1 + (ii / d3) % d2) * d2_scale + (ii / d3) % d2 * scale_factor + i) * d3_scale + (ii % d3) * scale_factor + j;
            gradInput_accum += gradOutput_data[ipidx];
        }
    }
    gradInput_data[ii] = gradInput_accum; // Assign once
}