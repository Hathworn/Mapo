#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_setup_curand(hiprandState *state, int seed, int n)
{
int id = threadIdx.x + blockIdx.x *  blockDim.x ;
/* Each thread gets same seed , a different sequence number - no offset */
if(id<n)
hiprand_init(seed, id, 0, &state[id]) ;
}