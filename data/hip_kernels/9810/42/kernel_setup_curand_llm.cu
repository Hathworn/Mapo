#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_setup_curand(hiprandState *state, int seed, int n)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    // Improved by launching only required threads
    if (id < n)
    {
        hiprand_init(seed, id, 0, &state[id]);
    }
}