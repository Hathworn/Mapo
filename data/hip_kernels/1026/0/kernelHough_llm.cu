#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define iceil(num, den) (num + den - 1) / den
#define ARRAY_SIZE 20
#define NUM_LINES 4

#define dimension 5
#define LXBOUND (-1*dimension)
#define RXBOUND (dimension)
#define LYBOUND (-1*dimension)
#define UYBOUND (dimension)

#define INCREMENT 1

#define column (((RXBOUND - LXBOUND) / INCREMENT) * ((RXBOUND - LXBOUND) / INCREMENT)) / ((RXBOUND + UYBOUND) / INCREMENT)

__constant__ int d_coordarray[ARRAY_SIZE];

__global__ void kernelHough(int size, int* d_binarray) {
    // Calculate unique thread index
    const int thread = 2 * (blockDim.x * blockIdx.x + threadIdx.x);

    // Access slope and intercept from constant memory
    const float slope = -1.0 * d_coordarray[thread];
    const float intercept = d_coordarray[thread + 1];

    // Initialize bin counter
    int counter = 0;

    // Loop through the discretized graph by bins
    for (float x = LXBOUND; x < RXBOUND; x += INCREMENT) {
        const float xMin = x;
        const float xMax = x + INCREMENT;

        for (float y = UYBOUND; y > LYBOUND; y -= INCREMENT) {
            const float yMin = y - INCREMENT;
            const float yMax = y;

            // Calculate potential lower and upper y ranges
            const float lower_range = slope * xMin + intercept;
            const float upper_range = slope * xMax + intercept;

            // Check if the calculated y ranges are within the actual bin range
            if ((lower_range <= yMax && lower_range >= yMin) || (upper_range <= yMax && upper_range >= yMin)) {
                // Update bin count with atomic operation to avoid race conditions
                atomicAdd(&d_binarray[counter], 1);
            }
            counter++;
        }
    }
}