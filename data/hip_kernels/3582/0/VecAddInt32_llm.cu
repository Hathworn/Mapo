#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAddInt32(int32_t* in0, int32_t* in1, int32_t* out, int cnt)
{
    // Calculate the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop for better memory access pattern
    for (int i = tid; i < cnt; i += blockDim.x * gridDim.x) {
        out[i] = in0[i] + in1[i];
    }
}