#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecSubInt32(int32_t* __restrict__ in0, int32_t* __restrict__ in1, int32_t* __restrict__ out, int cnt)
{
    // Using shared memory to reduce global memory access times
    extern __shared__ int32_t shared_in0[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    // Load current block's data into shared memory
    if (tid < cnt) {
        shared_in0[local_tid] = in0[tid];
        __syncthreads();

        // Perform subtraction using shared memory
        out[tid] = shared_in0[local_tid] - in1[tid];
    }
}