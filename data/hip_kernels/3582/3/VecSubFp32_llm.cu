#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecSubFp32(float* __restrict__ in0, float* __restrict__ in1, float* __restrict__ out, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensures the thread index is within bounds
    if (tid < cnt) {
        // Perform vector subtraction
        out[tid] = in0[tid] - in1[tid];
    }
}
```
