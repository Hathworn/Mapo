#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAddFp32(float* in0, float* in1, float* out, int cnt)
{
    // Improved access pattern for performance
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = blockDim.x * gridDim.x;
    
    for (int i = tid; i < cnt; i += totalThreads) {
        out[i] = in0[i] + in1[i];
    }
}