#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ptr2ind_kernel(const int64_t *ptr_data, int64_t *out_data, int64_t E, int64_t numel) {
    int64_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure thread_idx and next_idx are within bounds
    if (thread_idx < numel && thread_idx + 1 < numel + 1) {
        int64_t idx = ptr_data[thread_idx], next_idx = ptr_data[thread_idx + 1];

        // Use a single loop to avoid unnecessary condition checks
        for (int64_t i = idx; i < next_idx; i++) {
            out_data[i] = thread_idx;
        }
    }
}