#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void non_diag_mask_kernel(const int64_t *row_data, const int64_t *col_data, bool *out_data, int64_t N, int64_t k, int64_t num_diag, int64_t numel) {
    
    int64_t thread_idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (thread_idx < numel) {
        int64_t r = row_data[thread_idx], c = col_data[thread_idx];
        int64_t index = thread_idx + ((k < 0) ? r + k : r);  // Compute base index

        if (k < 0) {
            if (r + k < 0 || r + k >= N) {
                out_data[thread_idx + num_diag] = true;  // Handle off-diagonal cases
            } else {
                out_data[index + ((r + k > c) ? 0 : 1)] = true;  // Optimize index update
            }
        } else {
            if (r + k >= N) {
                out_data[thread_idx + num_diag] = true;
            } else {
                out_data[index + ((r + k > c) ? 0 : 1)] = true;
            }
        }
    }
}