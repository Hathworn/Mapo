#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 1024
#define TIME 3600000

__global__ void initialize(float *a_d, float *b_d, float *c_d, int arraySize)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;

    // Reduced the number of conditions evaluated
    if (ix < arraySize)
    {
        if (ix == 0)
        {
            a_d[ix] = 200.0f;  // Initialize first element with 200.0
            b_d[ix] = 200.0f;  // Initialize first element with 200.0
        }
        else
        {
            a_d[ix] = 0.0f;    // Initialize remaining elements with 0.0
            b_d[ix] = 0.0f;    // Initialize remaining elements with 0.0
        }
    }
}