#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 1024
#define TIME 3600000

__global__ void compute(float *a_d, float *b_d, float *c_d, int arraySize)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if (ix > 0 && ix < arraySize-1) {
        float temp = (a_d[ix+1] + a_d[ix-1]) / 2.0f; // Compute average of neighboring elements
        b_d[ix] = temp; // Store result in b_d
    }
}