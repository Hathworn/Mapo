#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prefixSum_UniqueBlock(float* in, int in_length, float* out ){

    //shared memory declaration
    extern __shared__ float DSM[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    //load in shared memory
    if(idx < in_length){
        DSM[threadIdx.x] = in[idx];
    } else {
        DSM[threadIdx.x] = 0.0f; // Handle out-of-bound threads
    }

    __syncthreads();

    //partial sums phase
    for(int stride = 1; stride < blockDim.x; stride *= 2) { // Change condition to '<'
        int index_aux = (threadIdx.x + 1) * stride * 2 - 1;
        if(index_aux < blockDim.x)
            DSM[index_aux] += DSM[index_aux - stride];
        __syncthreads();  // Synchronize here instead of outside the loop
    }

    //reduction phase
    for(int stride = blockDim.x / 4; stride > 0; stride /= 2) {
        int index_aux = (threadIdx.x + 1) * stride * 2 - 1;
        if(index_aux + stride < blockDim.x)
            DSM[index_aux + stride] += DSM[index_aux];
        __syncthreads();  // Synchronize here instead of outside the loop
    }

    if(idx < in_length) {
        out[idx] = DSM[threadIdx.x];
    }
}