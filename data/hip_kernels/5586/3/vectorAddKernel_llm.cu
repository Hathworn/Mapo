#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for vector addition
__global__ void vectorAddKernel(float* inputA, float* inputB, float* output, int length){
    // Compute element index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Add vector element using shared memory for better access speed
    extern __shared__ float sdata[];
    if (idx < length) {
        sdata[threadIdx.x] = inputA[idx] + inputB[idx];
        output[idx] = sdata[threadIdx.x];
    }
}