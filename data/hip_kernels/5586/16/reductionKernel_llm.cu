#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reductionKernel(float* vec, int width, double* sumUp){

    // shared memory instantiation
    extern __shared__ float partialSum[];

    // index for global memory
    int g_idx = blockDim.x * blockIdx.x + threadIdx.x;
    // index for shared memory
    int b_idx = threadIdx.x;

    // load shared memory from global memory
    partialSum[b_idx] = (g_idx < width) ? vec[g_idx] : 0;
    __syncthreads();  // Ensure all loads are complete

    // reduction inside blocks
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (b_idx < stride) {
            partialSum[b_idx] += partialSum[b_idx + stride];
        }
        __syncthreads();  // Synchronize to ensure all additions are complete
    }

    // reduction for grid using thread 0 of each block
    if (b_idx == 0) {
        // copy value back to global memory
        vec[blockIdx.x] = partialSum[0];
    }

    // Block 0 performs the final reduction across all blocks
    if (g_idx == 0) {
        double finalSum = 0.0;
        for (int i = 0; i < gridDim.x; ++i) {
            finalSum += vec[i];
        }
        *sumUp = finalSum;
    }
}