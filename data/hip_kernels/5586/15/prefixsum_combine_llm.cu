#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prefixsum_combine(float* in, int in_length, float* out, int out_length) {

    // Use shared memory to reduce global memory loads
    extern __shared__ float shared[];
    
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;

    // Load data to shared memory
    if (tid < in_length) {
        shared[tid] = in[tid];
    }
    __syncthreads();
    
    // Only proceed if within bounds
    if (idx < out_length && blockIdx.x > 0) {
        out[idx] += shared[blockIdx.x - 1]; // Reduce global memory access
    }
}