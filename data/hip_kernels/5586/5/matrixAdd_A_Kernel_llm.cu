#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixAdd_A_Kernel(float* A, float* B, float* C, size_t pitch, int width) {

    // Using shared memory for better locality and reduced global memory access latency
    __shared__ float sA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float sB[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int rowWidthWithPad = pitch / sizeof(float);

    // Load data from global memory to shared memory
    if (row < width && col < width) {
        sA[threadIdx.x][threadIdx.y] = A[row * rowWidthWithPad + col];
        sB[threadIdx.x][threadIdx.y] = B[row * rowWidthWithPad + col];
    }
    __syncthreads();  // Ensure all data is loaded before proceeding

    // Only proceed if within bounds
    if (row < width && col < width) {
        // Compute and store into global memory
        C[row * rowWidthWithPad + col] = sA[threadIdx.x][threadIdx.y] + sB[threadIdx.x][threadIdx.y];
    }
}