#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hist_inGlobal(const int* values, int length, int* hist) {
    // Calculate global thread index
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Calculate stride to handle more elements per thread
    int stride = blockDim.x * gridDim.x;

    // Use shared memory to cache histogram updates
    __shared__ int local_hist[256]; // Assume 'values' range from 0 to 255

    // Initialize shared memory
    if (threadIdx.x < 256) {
        local_hist[threadIdx.x] = 0;
    }
    __syncthreads();

    // Iterate with stride
    while (idx < length) {
        int val = values[idx];
        // Use atomic operations in shared memory
        atomicAdd(&local_hist[val], 1);
        idx += stride;
    }
    __syncthreads();

    // Move shared memory results to global memory
    if (threadIdx.x < 256) {
        atomicAdd(&hist[threadIdx.x], local_hist[threadIdx.x]);
    }
}