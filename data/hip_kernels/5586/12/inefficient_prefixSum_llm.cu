#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void efficient_prefixSum(float* in, int in_length, float* out) {
    // Shared memory declaration
    extern __shared__ float DSM[];

    // Compute index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < in_length) {
        // Load into shared memory
        DSM[threadIdx.x] = in[idx];
        __syncthreads(); // Ensure all data is loaded into shared memory

        // Compute prefix_sum using a more efficient approach
        for (int stride = 1; stride < blockDim.x; stride *= 2) {
            float temp = 0.0f;
            if (threadIdx.x >= stride) {
                temp = DSM[threadIdx.x - stride];
            }
            __syncthreads();
            DSM[threadIdx.x] += temp;
            __syncthreads();
        }

        out[idx] = DSM[threadIdx.x];
    }
}