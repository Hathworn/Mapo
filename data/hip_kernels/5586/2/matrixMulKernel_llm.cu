#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMulKernel(float* d_M, float* d_N, float* d_P, int width) {

    // Compute row and column of the target element to compute
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure row and column are within bounds
    if (row < width && col < width) {
        float sum = 0.0f;

        // Optimize memory access by using shared memory for matrix tiles
        __shared__ float M_shared[32][32];
        __shared__ float N_shared[32][32];

        // Iterate over tiles
        for (int tile = 0; tile < width / 32; ++tile) {
            
            // Load tiles into shared memory
            M_shared[threadIdx.y][threadIdx.x] = d_M[row * width + tile * 32 + threadIdx.x];
            N_shared[threadIdx.y][threadIdx.x] = d_N[(tile * 32 + threadIdx.y) * width + col];
            __syncthreads(); // Synchronize to ensure all threads have loaded their data

            // Compute partial dot product
            for (int k = 0; k < 32; ++k) {
                sum += M_shared[threadIdx.y][k] * N_shared[k][threadIdx.x];
            }
            __syncthreads(); // Synchronize before loading the next tile
        }

        // Assign computed value
        d_P[row * width + col] = sum;
    }
}