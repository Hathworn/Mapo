#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prefixSum_multiBlocks(float* in, int in_length, float* out, float* temp) {

    extern __shared__ float DSM[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input into shared memory
    if (idx < in_length) {
        DSM[threadIdx.x] = in[idx];
    } else {
        DSM[threadIdx.x] = 0.0f; // Ensures all threads read initialized values
    }

    // Partial sums phase
    for (int stride = 1; stride <= blockDim.x; stride *= 2) {
        __syncthreads();
        int index_aux = (threadIdx.x + 1) * 2 * stride - 1;
        if (index_aux < blockDim.x) {
            DSM[index_aux] += DSM[index_aux - stride];
        }
    }

    // Reduction phase
    for (int stride = blockDim.x / 4; stride > 0; stride /= 2) {
        __syncthreads();
        int index_aux = (threadIdx.x + 1) * 2 * stride - 1;
        if (index_aux + stride < blockDim.x) {
            DSM[index_aux + stride] += DSM[index_aux];
        }
    }

    // Ensure all calculations are done before using DSM values
    __syncthreads();

    // Save the last element in each block to temp for post-combine operation
    if (threadIdx.x == 0) {
        temp[blockIdx.x] = DSM[blockDim.x - 1];
    }

    // Write results to output
    if (idx < in_length) {
        out[idx] = DSM[threadIdx.x];
    }
}