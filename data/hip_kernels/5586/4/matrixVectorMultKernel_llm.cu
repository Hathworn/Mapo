#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixVectorMultKernel(float* fltMatrix, float* vec, float* output, int rows, int columns) {
    int row = blockDim.x * blockIdx.x + threadIdx.x;
    // Check if within valid row range
    if(row < rows) {
        float sum = 0.0f;
        // Use shared memory for vector to optimize access
        extern __shared__ float sharedVec[];
        if (threadIdx.x < columns) {
            sharedVec[threadIdx.x] = vec[threadIdx.x];
        }
        __syncthreads();
        
        // Perform dot product
        for (int col = 0; col < columns; ++col) {
            sum += fltMatrix[row * columns + col] * sharedVec[col];
        }
        output[row] = sum;
    }
}