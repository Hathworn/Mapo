#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixAdd_B_Kernel(float* A, float* B, float* C, size_t pitch, int width) {
    // Compute indexes
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;  // Use blockIdx.y and threadIdx.y for col index
    int rowWidthWithPad = pitch / sizeof(float);

    // Add boundary check for both row and col
    if (row < width && col < width) {
        C[row * rowWidthWithPad + col] = A[row * rowWidthWithPad + col] + B[row * rowWidthWithPad + col];
    }
}