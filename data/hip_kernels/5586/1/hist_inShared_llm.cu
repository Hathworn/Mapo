#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//kernel for computing histogram right in memory

//compute partial histogram on shared memory and mix them on global memory

__global__ void hist_inShared (const int* values, int length, int* hist) {

//load shared memory
extern __shared__ int shHist[];

// Each thread block initializes its own shared memory histogram to 0
shHist[threadIdx.x] = 0;
__syncthreads();

// Compute global index and stride for this thread
int idx = blockDim.x * blockIdx.x + threadIdx.x;
int stride = gridDim.x * blockDim.x;

// Iterate over all values within bounds that this thread should process
while (idx < length) {
    int val = values[idx];

    // Increment value frequency on shared histogram
    atomicAdd(&shHist[val], 1);

    // Move to the next value that this thread should process
    idx += stride;
}

// Wait for all threads to finish updating shared histogram
__syncthreads();

// Combine partial histogram on shared memory to create a full histogram
atomicAdd(&hist[threadIdx.x], shHist[threadIdx.x]);
}