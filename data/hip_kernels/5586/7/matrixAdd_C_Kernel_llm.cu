#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixAdd_C_Kernel(float* A, float* B, float* C, size_t pitch, int width) {
    // Compute indexes
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int rowWidthWithPad = pitch / sizeof(float);

    // Check bounds for valid threads
    if (col < width && row < width) {
        // Directly calculate the index for each element
        int index = row * rowWidthWithPad + col;
        C[index] = A[index] + B[index];
    }
}