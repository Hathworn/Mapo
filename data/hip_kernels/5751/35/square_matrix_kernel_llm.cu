#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Compute linear index for row and column
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread operates within matrix dimensions
    if (row < num_rows && col < num_cols) {
        float f = feats[row * ldf + col];
        feats_sq[row * lds + col] = f * f;
    }
}