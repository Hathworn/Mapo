#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    float *feats = features + frame * ldf;

    // Apply lifter coefficients with optimized memory coalescing
    if (cepstral_lifter != 0.0f) {
        for (int c = thread_id; c < num_cols; c += blockDim.x) { // Use blockDim.x instead of CU1DBLOCK
            float lift = lifter_coeffs[c];
            atomicExch(&feats[c], feats[c] * lift); // Apply atomic operation for synchronization
        }
    }

    // Thread 0 for each frame will apply energy
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = logf(energy_floor); // Use logf for single precision

        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        atomicExch(&feats[0], energy); // Use atomic operation to avoid race conditions
    }
}