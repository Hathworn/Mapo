#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel(int32_t n, float old_num_frames, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    float cur_weight = *cur_tot_weight;
    
    float new_num_frames = old_num_frames + cur_weight;
    float prior_scale_change = 1.0f;

    if (max_count != 0.0f) {
        float old_prior_scale = max(old_num_frames, (float)max_count) / max_count;
        float new_prior_scale = max(new_num_frames, (float)max_count) / max_count;
        prior_scale_change += new_prior_scale - old_prior_scale;
    }

    int32_t start_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride = blockDim.x * gridDim.x;

    // Iterate using flattened indices for better thread utilization
    for (int32_t i = start_idx; i < n; i += stride) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        quadratic[diag_idx] += prior_scale_change;
    }

    // Use a single thread after looping to do the linear update
    if (start_idx == 0) {
        linear[0] += prior_offset * prior_scale_change;
    }
}