#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;

    // Optimize loop using shared memory
    extern __shared__ float shared_data[];
    float *shared_Ar = shared_data;

    if (thread_id < half_length) {
        shared_Ar[thread_id] = Ar[thread_id];
    }
    __syncthreads();

    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {
        // Ignore special case
        if (idx == 0) continue;

        // Use shared memory for data access
        float2 val = reinterpret_cast<float2 *>(shared_Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;

        if (use_power) {
            Aw[idx] = ret;
        } else {
            Aw[idx] = sqrtf(ret);
        }
    }

    // Handle special case using first thread
    if (thread_id == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        if (use_power) {
            Aw[0] = real * real;
            Aw[half_length] = im * im;
        } else {
            Aw[0] = fabs(real);
            Aw[half_length] = fabs(im);
        }
    }
}