#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SimpleClone(const float *background, const float *target, const float *mask, float *output, const int wb, const int hb, const int wt, const int ht, const int oy, const int ox)
{
    // Compute global thread indices
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Guard check indices
    if (yt < ht && xt < wt) {
        const int curt = wt * yt + xt;
        if (mask[curt] > 127.0f) {
            const int yb = oy + yt;
            const int xb = ox + xt;
            
            // Bound check for indices
            if (yb >= 0 && yb < hb && xb >= 0 && xb < wb) {
                const int curb = wb * yb + xb;

                // Direct copy of target pixel to output
                for (int i = 0; i < 3; ++i) {
                    output[curb * 3 + i] = target[curt * 3 + i];
                }
            }
        }
    }
}