#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Perform division with atomic operation to prevent race conditions
    atomicExch(accuracy, *accuracy / N);
}