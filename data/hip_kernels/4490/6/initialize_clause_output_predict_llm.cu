#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void initialize_clause_output_predict(int *clause_output, int *all_exclude)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use single condition to avoid branch divergence
    if(index < CLAUSES) {
        // Each thread initializes its own clause output
        clause_output[index] = 1;
        all_exclude[index] = 1;
    }
}