#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_with_all_exclude(int *clause_output, int *all_exclude)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Use shared memory to reduce global memory access
    __shared__ int shared_exclude[CLAUSES];

    // Load data into shared memory
    for (int j = index; j < CLAUSES; j += stride) {
        shared_exclude[j] = all_exclude[j];
    }
    __syncthreads();

    // Update clause output with shared memory
    for (int j = index; j < CLAUSES; j += stride) {
        if (shared_exclude[j] == 1) {
            clause_output[j] = 0;
        }
    }
}