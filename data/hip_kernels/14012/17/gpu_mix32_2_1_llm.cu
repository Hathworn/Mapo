#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_mix32_2_1(int64_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples)
{
    int z = blockDim.x * blockIdx.x + threadIdx.x;

    // Early exit for out-of-bound threads
    if (z >= numSamples)
        return;

    int64_t temp = ip[z];

    // Directly assign to minimize conversion repetition
    u[z] = static_cast<int32_t>(temp);
    v[z] = static_cast<int32_t>(temp >> 32);
}