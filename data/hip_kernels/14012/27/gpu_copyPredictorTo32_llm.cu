#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_copyPredictorTo32(int32_t * in, int32_t * out, uint32_t stride, uint32_t * numSamples, int32_t theOutputPacketBytes, uint32_t frameLength)
{
    int block = blockIdx.x % 8;
    int index = blockIdx.x / 8;
    int z = threadIdx.x + block * blockDim.x;

    if (z < numSamples[index])
    {
        int32_t * op = out + (index * theOutputPacketBytes) / 4;

        // Coalesced memory access by using only one pointer arithmetic
        int32_t * inputBase = in + index * frameLength; // Pre-compute input base pointer
        op[stride * z] = inputBase[z];
    }
}