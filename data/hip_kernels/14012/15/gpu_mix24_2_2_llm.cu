#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_mix24_2_2(uint8_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples)
{
    int z = threadIdx.x + blockIdx.x * blockDim.x;
    if (z < numSamples)
    {
        int32_t l, r;
        uint8_t* ip_offset = ip + 3 * z * stride;  // Calculate offset once

        // Load and process left sample
        l = (int32_t)(((uint32_t)ip_offset[HBYTE] << 16) | ((uint32_t)ip_offset[MBYTE] << 8) | (uint32_t)ip_offset[LBYTE]);
        u[z] = (l << 8) >> 8;
        
        // Load and process right sample
        ip_offset += 3;  // Increment offset directly
        r = (int32_t)(((uint32_t)ip_offset[HBYTE] << 16) | ((uint32_t)ip_offset[MBYTE] << 8) | (uint32_t)ip_offset[LBYTE]);
        v[z] = (r << 8) >> 8;
    }
}