#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_mix24_1_2(uint8_t *ip, uint32_t stride, int32_t *u, int32_t *v, int32_t numSamples, int32_t mixres, int32_t m2, int32_t mixbits)
{
    int z = threadIdx.x + blockIdx.x * blockDim.x;
    if (z < numSamples)
    {
        // Pointer arithmetic optimization
        uint8_t *ip_z = ip + 3 * ((stride - 1) * z + z);
        
        // Use local variables to avoid repeated pointer dereferencing
        uint32_t l_high = (uint32_t)ip_z[HBYTE];
        uint32_t l_middle = (uint32_t)ip_z[MBYTE];
        uint32_t l_low = (uint32_t)ip_z[LBYTE];

        int32_t l = (int32_t)((l_high << 16) | (l_middle << 8) | l_low);
        l = (l << 8) >> 8; // Byte re-sign extension

        ip_z += 3;
        uint32_t r_high = (uint32_t)ip_z[HBYTE];
        uint32_t r_middle = (uint32_t)ip_z[MBYTE];
        uint32_t r_low = (uint32_t)ip_z[LBYTE];

        int32_t r = (int32_t)((r_high << 16) | (r_middle << 8) | r_low);
        r = (r << 8) >> 8; // Byte re-sign extension

        u[z] = (mixres * l + m2 * r) >> mixbits; // Mixing operation
        v[z] = l - r; // Difference calculation
    }
}