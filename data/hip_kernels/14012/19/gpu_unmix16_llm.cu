#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_unmix16(int32_t * u, int32_t * v, int16_t * out, uint32_t stride, uint32_t * numSamples, int32_t * mixbits, int32_t * mixres, int32_t theOutputPacketBytes, uint32_t frameLength)
{
    int block = blockIdx.x % 8;
    int index = blockIdx.x / 8;
    int z = threadIdx.x + block * blockDim.x;

    if (z < numSamples[index]) 
    {
        int32_t l, r;
        int16_t *op = out + (index * theOutputPacketBytes) / 2 + stride * z;  // Move stride calculation outside condition

        int32_t u_val = (u + index * frameLength)[z];  // Load shared values to local variables
        int32_t v_val = (v + index * frameLength)[z];

        if (mixres[index] != 0)
        {
            // Matrixed stereo processing
            l = u_val + v_val - ((mixres[index] * v_val) >> mixbits[index]);
            r = l - v_val;
        }
        else
        {
            // Conventional separated stereo
            l = u_val;
            r = v_val;
        }
        
        op[0] = (int16_t)l;  // Write computed values
        op[1] = (int16_t)r;
    }
}