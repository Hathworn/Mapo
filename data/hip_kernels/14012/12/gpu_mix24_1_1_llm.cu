#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_mix24_1_1(uint8_t *ip, uint32_t stride, int32_t *u, int32_t *v, int32_t numSamples, uint16_t *shiftUV, int32_t mixres, uint32_t mask, int32_t m2, int32_t mixbits, int32_t shift)
{
    int z = blockIdx.x * blockDim.x + threadIdx.x;
    if (z >= numSamples) return; // Simplified boundary check

    int32_t l, r;
    int32_t k = z * 2;

    // Efficiently calculate the base pointer for `ip`
    uint8_t *ipCurrent = ip + (3 * stride - 3) * z;

    // Merge the extraction and conversion of l and r
    l = (((int32_t)ipCurrent[2] << 16) | ((int32_t)ipCurrent[1] << 8) | (int32_t)ipCurrent[0]) << 8 >> 8;
    ipCurrent += 3;
    r = (((int32_t)ipCurrent[2] << 16) | ((int32_t)ipCurrent[1] << 8) | (int32_t)ipCurrent[0]) << 8 >> 8;

    // Update shiftUV array
    shiftUV[k] = (uint16_t)(l & mask);
    shiftUV[k + 1] = (uint16_t)(r & mask);

    // Shifting l and r
    l >>= shift;
    r >>= shift;

    // Updating u and v
    u[z] = (mixres * l + m2 * r) >> mixbits;
    v[z] = l - r;
}