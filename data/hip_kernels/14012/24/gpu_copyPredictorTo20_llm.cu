#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// HBYTE, MBYTE, LBYTE indices are assumed to be pre-defined or replaced with actual values
__global__ void gpu_copyPredictorTo20(int32_t * in, uint8_t * out, uint32_t stride, uint32_t * numSamples, int32_t theOutputPacketBytes, uint32_t frameLength)
{
    int block = blockIdx.x % 8;
    int index = blockIdx.x / 8;
    int z = threadIdx.x + block * blockDim.x;
    uint8_t *op;

    if (z < numSamples[index]) 
    {
        op = out + index * theOutputPacketBytes + stride * 3 * z; // Calculate op offset once

        // Load value and extract bytes directly
        int32_t val = (in + index * frameLength)[z];
        op[HBYTE] = (uint8_t)((val >> 12) & 0xffu); // Extract and assign bytes
        op[MBYTE] = (uint8_t)((val >> 4) & 0xffu);
        op[LBYTE] = (uint8_t)((val << 4) & 0xffu);
    }
}