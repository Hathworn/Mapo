#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_copyPredictorTo16(int32_t *in, int16_t *out, uint32_t stride, uint32_t *numSamples, int32_t theOutputPacketBytes, uint32_t frameLength)
{
    // Improved memory coalescing by calculating block and index using blockIdx.x and blockIdx.y
    int block = threadIdx.x + (blockIdx.x % 8 * blockDim.x);
    int index = blockIdx.x / 8;
    
    if (block < numSamples[index])
    {
        int16_t *op = out + (index * theOutputPacketBytes) / 2;
        
        // Efficient access pattern for global memory
        op[block] = static_cast<int16_t>(in[index * frameLength + block]);
    }
}