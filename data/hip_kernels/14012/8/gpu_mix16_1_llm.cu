#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_mix16_1(int32_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples, int32_t m2, int32_t mixbits, int32_t mixres) {
    int z = threadIdx.x + blockIdx.x * blockDim.x;

    if (z < numSamples) {
        // Unroll and simplify operations for better performance
        int32_t temp = ip[z];
        int16_t l = (int16_t)temp;
        int16_t r = (int16_t)(temp >> 16);

        // Use a single multiplication for vector optimization
        int32_t mixed = (mixres * l + m2 * r);

        // Reduce shifts and use pre-computation when possible
        u[z] = mixed >> mixbits;
        v[z] = l - r;
    }
}