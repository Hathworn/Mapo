#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_unmix32(int32_t * u, int32_t * v, int32_t * out, uint32_t stride, uint32_t * numSamples, int32_t * mixbits, int32_t * mixres, uint16_t * shiftUV, int32_t bytesShifted, int32_t theOutputPacketBytes, uint32_t frameLength)
{
    // Precompute shared values to reduce redundant calculations
    int32_t shift = bytesShifted * 8;
    int block = blockIdx.x % 8;
    int index = blockIdx.x / 8;
    int z = threadIdx.x + block * blockDim.x;
    if (z >= numSamples[index]) return; // Eliminate extra conditions

    int32_t * op = out + (index * theOutputPacketBytes) / 4 + stride * z;
    int32_t k = z * 2;
    int32_t lt = (u + index * frameLength)[z];
    int32_t rt = (v + index * frameLength)[z];
    uint16_t* shiftUV_ptr = shiftUV + index * frameLength * 2 + k; // Precompute address

    if (mixres[index] != 0)
    {
        // Simplify calculations by reducing operations
        int32_t l = lt + rt - ((mixres[index] * rt) >> mixbits[index]);
        int32_t r = l - rt;

        op[0] = (l << shift) | (uint32_t)shiftUV_ptr[0];
        op[1] = (r << shift) | (uint32_t)shiftUV_ptr[1];
    }
    else
    {
        op[0] = (lt << shift) | (uint32_t)shiftUV_ptr[0];
        op[1] = (rt << shift) | (uint32_t)shiftUV_ptr[1];
    }
}