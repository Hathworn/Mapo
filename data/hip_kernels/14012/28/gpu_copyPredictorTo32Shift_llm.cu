#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_copyPredictorTo32Shift(int32_t * in, uint16_t * shift, int32_t * out, uint32_t stride, uint32_t * numSamples, int32_t bytesShifted, int32_t theOutputPacketBytes, uint32_t frameLength)
{
    int block = blockIdx.x % 8;
    int index = blockIdx.x / 8;
    int z = threadIdx.x + block * blockDim.x;

    // Precompute commonly used values outside condition block
    int32_t shiftVal = bytesShifted * 8;
    int32_t * op = out + (index * theOutputPacketBytes) / 4 + stride * z;
    
    if (z < numSamples[index])
    {
        // Use local variable to reduce redundant calculations
        const int32_t * in_offset = in + index * frameLength;
        const uint16_t * shift_offset = shift + index * frameLength * 2;
        
        op[0] = (in_offset[z] << shiftVal) | (uint32_t)shift_offset[z];
    }
}