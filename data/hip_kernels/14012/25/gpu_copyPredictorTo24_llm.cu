#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_copyPredictorTo24(int32_t *in, uint8_t *out, uint32_t stride, uint32_t *numSamples, int32_t theOutputPacketBytes, uint32_t frameLength)
{
    // Compute global thread index and restrict memory access within bounds
    int index = blockIdx.x / 8;
    int z = threadIdx.x + (blockIdx.x % 8) * blockDim.x;

    if (z < numSamples[index]) {
        // Compute output pointer offset directly from global index
        uint8_t *op = out + index * theOutputPacketBytes + stride * 3 * z;
        int32_t val = in[index * frameLength + z];

        // Efficient byte extraction using bit shifting
        op[HBYTE] = (uint8_t)((val >> 16) & 0xffu);
        op[MBYTE] = (uint8_t)((val >> 8) & 0xffu);
        op[LBYTE] = (uint8_t)(val & 0xffu);
    }
}