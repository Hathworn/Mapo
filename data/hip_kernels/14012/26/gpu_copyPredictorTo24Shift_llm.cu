#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_copyPredictorTo24Shift(int32_t * in, uint16_t * shift, uint8_t * out, uint32_t stride, uint32_t * numSamples, int32_t bytesShifted, int32_t theOutputPacketBytes, uint32_t frameLength)
{
    // Calculate a single linear thread index for optimizing memory access patterns
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int index = idx / (8 * blockDim.x);  // Calculate index from linear idx
    int block = idx % (8 * blockDim.x) / blockDim.x;  // Calculate block from linear idx
    int z = idx % blockDim.x + block * blockDim.x;

    if (z < numSamples[index])
    {
        uint8_t * op = out + (index * theOutputPacketBytes);
        int32_t shiftVal = bytesShifted * 8;

        int32_t val = (in + index * frameLength)[z];
        val = (val << shiftVal) | (uint32_t)(shift + index * frameLength * 2)[z];

        op += (stride * 3 * z);
        op[HBYTE] = (uint8_t)((val >> 16) & 0xffu);
        op[MBYTE] = (uint8_t)((val >> 8) & 0xffu);
        op[LBYTE] = (uint8_t)((val >> 0) & 0xffu);
    }
}