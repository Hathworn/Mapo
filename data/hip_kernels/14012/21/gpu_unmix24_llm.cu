#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Define macros for byte offsets
#define HBYTE 0
#define MBYTE 1
#define LBYTE 2

__global__ void gpu_unmix24(int32_t *u, int32_t *v, uint8_t *out, uint32_t stride, uint32_t *numSamples, int32_t *mixbits, int32_t *mixres, uint16_t *shiftUV, int32_t bytesShifted, int32_t theOutputPacketBytes, uint32_t frameLength)
{
    int block = blockIdx.x % 8;
    int index = blockIdx.x / 8;
    int z = threadIdx.x + block * blockDim.x;

    if (z < numSamples[index])
    {
        int32_t shift = bytesShifted * 8;
        int32_t l, r;
        int32_t k = z * 2;
        uint8_t *op = out + (index * theOutputPacketBytes);
        int32_t uVal = (u + index * frameLength)[z];
        int32_t vVal = (v + index * frameLength)[z];
        uint16_t shiftUVValue1 = (shiftUV + index * frameLength * 2)[k + 0];
        uint16_t shiftUVValue2 = (shiftUV + index * frameLength * 2)[k + 1];

        if (mixres[index] != 0)
        {
            // optimized calculation for matrixed stereo
            int32_t temp = (mixres[index] * vVal) >> mixbits[index];
            l = uVal + vVal - temp;
            r = l - vVal;
        }
        else
        {
            // optimized calculation for conventional separated stereo
            l = uVal;
            r = vVal;
        }

        l = (l << shift) | shiftUVValue1;
        r = (r << shift) | shiftUVValue2;

        // Use efficient memory addressing and assignments
        op += (stride + 1) * 3 * z; 
        op[HBYTE] = (uint8_t)((l >> 16) & 0xffu);
        op[MBYTE] = (uint8_t)((l >> 8) & 0xffu);
        op[LBYTE] = (uint8_t)((l >> 0) & 0xffu);
        op += 3;

        op[HBYTE] = (uint8_t)((r >> 16) & 0xffu);
        op[MBYTE] = (uint8_t)((r >> 8) & 0xffu);
        op[LBYTE] = (uint8_t)((r >> 0) & 0xffu);
    }
}