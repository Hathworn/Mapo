#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_mix32_2_2(int64_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples, uint16_t * shiftUV, uint32_t mask, int32_t shift)
{
    int z = threadIdx.x + blockIdx.x * blockDim.x;
    // Ensure that only valid threads work
    if (z < numSamples)
    {
        int32_t l, r;
        int32_t k = z * 2;
        int64_t temp = ip[z];

        // Reduce overflow risk by masking first
        shiftUV[k] = (uint16_t)(temp & mask); // Optimize indexing with single access
        shiftUV[k + 1] = (uint16_t)((temp >> 32) & mask);

        // Perform bit-shift operation after masking
        u[z] = (int32_t)(temp >> shift);
        v[z] = (int32_t)((temp >> 32) >> shift);
    }
}