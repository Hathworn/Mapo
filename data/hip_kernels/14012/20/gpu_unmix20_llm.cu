#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_unmix20(int32_t * u, int32_t * v, uint8_t * out, uint32_t stride, uint32_t * numSamples, int32_t * mixbits, int32_t * mixres, int32_t theOutputPacketBytes, uint32_t frameLength)
{
    // Calculate effective thread index for processing
    int myIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int index = myIndex / blockDim.x;  // Specific output packet index

    // Check if thread is within number of samples
    if (threadIdx.x < numSamples[index])
    {
        int32_t l, r;
        uint8_t * op = out + (index * theOutputPacketBytes);

        int32_t u_val = (u + index * frameLength)[threadIdx.x];
        int32_t v_val = (v + index * frameLength)[threadIdx.x];
        
        // Matrixed stereo mode
        if (mixres[index] != 0)
        {
            l = u_val + v_val - ((mixres[index] * v_val) >> mixbits[index]);
            r = l - v_val;
        }
        else
        {
            // Conventional separated stereo mode
            l = u_val;
            r = v_val;
        }

        // Left and Right channel packing and storing
        l <<= 4;
        r <<= 4;

        op += 3 * threadIdx.x + (stride - 1) * 3 * threadIdx.x;
        op[HBYTE] = (uint8_t)((l >> 16) & 0xffu);
        op[MBYTE] = (uint8_t)((l >> 8) & 0xffu);
        op[LBYTE] = (uint8_t)((l >> 0) & 0xffu);
        op += 3;

        op[HBYTE] = (uint8_t)((r >> 16) & 0xffu);
        op[MBYTE] = (uint8_t)((r >> 8) & 0xffu);
        op[LBYTE] = (uint8_t)((r >> 0) & 0xffu);
    }
}