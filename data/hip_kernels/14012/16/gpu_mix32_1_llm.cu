#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_mix32_1(int64_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples, uint16_t * shiftUV, int32_t mixres, uint32_t mask, int32_t m2, int32_t mixbits, int32_t shift)
{
    int z = threadIdx.x + blockIdx.x * blockDim.x;
    if (z < numSamples)
    {
        int32_t l, r;
        int32_t k = z * 2;

        // Fetch input more efficiently
        int64_t temp = ip[z * stride]; 

        // Split into l and r
        l = (int32_t)temp;
        r = (int32_t)(temp >> 32);

        // Capture shiftUV with direct application of mask
        shiftUV[k] = (uint16_t)(l & mask);
        shiftUV[k + 1] = (uint16_t)(r & mask);

        // Pre-calculate shifts
        l >>= shift;
        r >>= shift;

        // Use fused multiply-add for computation
        u[z] = __fma_rn(mixres, l, m2 * r) >> mixbits;
        v[z] = l - r;
    }
}