#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void call_kALACSearch(int16_t * mCoefsU, int16_t * mCoefsV, int32_t kALACMaxCoefs)
{
    int x = blockIdx.x;
    int y = threadIdx.x;
    int index = x * 16 * 16 + y * 16;
    int32_t den = 1 << DENSHIFT_DEFAULT;

    // Pre-calculate shift division
    int32_t ainit_den = (AINIT * den) >> 4;
    int32_t binit_den = (BINIT * den) >> 4;
    int32_t cinit_den = (CINIT * den) >> 4;

    // Unroll initialization for U and V
    mCoefsU[index + 0] = ainit_den;
    mCoefsU[index + 1] = binit_den;
    mCoefsU[index + 2] = cinit_den;

    mCoefsV[index + 0] = ainit_den;
    mCoefsV[index + 1] = binit_den;
    mCoefsV[index + 2] = cinit_den;
    
    // Merged loop for setting coefficients to zero
    #pragma unroll
    for (int32_t k = 3; k < kALACMaxCoefs; k++)
    {
        mCoefsU[index + k] = 0;
        mCoefsV[index + k] = 0;
    }
}