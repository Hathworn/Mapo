#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_mix24_2_1(uint8_t * ip, uint32_t stride, int32_t * u, int32_t * v, int32_t numSamples, uint16_t * shiftUV, uint32_t mask, int32_t shift)
{
    int z = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure thread index is within bounds
    if (z < numSamples)
    {
        int32_t l, r;
        int32_t k = z * 2;

        // Optimize memory access by calculating offsets only once
        uint8_t* ip_base = ip + (stride * z * 3);
        uint8_t* ip_offset = ip_base + z * 3;

        // Pack bytes into 32-bit integer efficiently
        l = (int32_t)(((uint32_t)ip_offset[HBYTE] << 16) | ((uint32_t)ip_offset[MBYTE] << 8) | ((uint32_t)ip_offset[LBYTE]));
        l = (l << 8) >> 8;  // Remove sign bit efficiently

        ip_offset += 3;
        r = (int32_t)(((uint32_t)ip_offset[HBYTE] << 16) | ((uint32_t)ip_offset[MBYTE] << 8) | ((uint32_t)ip_offset[LBYTE]));
        r = (r << 8) >> 8;  // Remove sign bit efficiently

        // Store results with mask application
        shiftUV[k] = (uint16_t)(l & mask);
        shiftUV[k + 1] = (uint16_t)(r & mask);

        // Calculate shifted values more efficiently
        u[z] = l >> shift;
        v[z] = r >> shift;
    }
}