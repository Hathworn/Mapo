#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;

    // Unroll the loop for better coalesced access
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        shmem[threadIdx.x] += a[eidx] * b[eidx];
    }

    __syncthreads();

    // Use optimized reduction using warp shuffle
    for (int offset = 256; offset > 0; offset /= 2) {
        if (threadIdx.x < offset) {
            shmem[threadIdx.x] += shmem[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Final reduction within a warp using warp meeting
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];

        // Store the result back to global memory
        if (threadIdx.x == 0) {
            target[blockIdx.x] = mysh[0];
        }
    }
}