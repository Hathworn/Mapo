#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Directly dividing accuracy using one thread to avoid race conditions
    if (threadIdx.x == 0 && blockIdx.x == 0) {  // Ensure only one thread performs division
        *accuracy /= N;
    }
}