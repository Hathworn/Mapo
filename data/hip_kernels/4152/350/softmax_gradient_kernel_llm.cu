#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    float tmp = 0.0f;

    // Accumulate inner products using a warp reduction
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }

    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Perform a warp reduction to accumulate results
    if (idx < 32) {
        tmp = reduction_buffer[idx];
        for (int offset = 32; offset < blockDim.x; offset += 32) {
            tmp += reduction_buffer[idx + offset];
        }
        reduction_buffer[idx] = tmp;
    }
    __syncthreads();

    // Compute gradient using the reduced inner product
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}