#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate the global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    
    // Loop over all elements with stride of total number of threads
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Calculate 2D indices from the linear index for target array
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;

        // Get corresponding source indices using modulo
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Assign value from source to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}