#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop to improve warp execution efficiency
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];
        float scaledX = loc.x * scale;
        float scaledY = loc.y * scale;

        x[ptidx] = scaledX;
        y[ptidx] = scaledY;
    }
}