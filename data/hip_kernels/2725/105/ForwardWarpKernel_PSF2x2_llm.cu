#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);

    int tx = (int)px;
    int ty = (int)py;

    float value = src[image_row_offset + j];
    float weight;

    // Consolidate boundary checks and atomic adds
    for (int x_offset = 0; x_offset <= 1; ++x_offset) {
        for (int y_offset = 0; y_offset <= 1; ++y_offset) {
            int ntx = tx - 1 + x_offset;
            int nty = ty - 1 + y_offset;
            if (ntx >= 0 && ntx < w && nty >= 0 && nty < h) {
                weight = ((x_offset == 0) ? (1.0f - dx) : dx) * ((y_offset == 0) ? (1.0f - dy) : dy);
                int index = nty * image_stride + ntx;
                _atomicAdd(dst + index, value * weight);
                _atomicAdd(normalization_factor + index, weight);
            }
        }
    }
}