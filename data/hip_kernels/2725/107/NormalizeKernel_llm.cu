#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Use shared memory for better memory access times
    extern __shared__ float shared_normalization_factor[];
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    if (i >= h || j >= w) return;
    
    // Load data into shared memory
    int pos = i * s + j;
    shared_normalization_factor[threadIdx.y * blockDim.x + threadIdx.x] = normalization_factor[pos];
    __syncthreads();

    float scale = shared_normalization_factor[threadIdx.y * blockDim.x + threadIdx.x];
    
    float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

    image[pos] *= invScale;
}