#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Calculate position for each thread
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop to process more data if thread count is less than data length
    while (pos < len) 
    {
        d_res[pos] = d_src[pos] * scale;
        // Advance index by total number of threads
        pos += blockDim.x * gridDim.x;
    }
}