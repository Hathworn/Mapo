#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel function to launch a single thread
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    if (threadIdx.x == 0 && blockIdx.x == 0) { // Ensure a single thread executes
        *accuracy /= N;
    }
}