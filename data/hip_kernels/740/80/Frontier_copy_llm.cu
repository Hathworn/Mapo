#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Frontier_copy(unsigned int *frontier, unsigned int *frontier2, unsigned int *frontier_length) {
    // Calculate global index for each thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Limit execution to frontier length
    if (idx < *frontier_length) {
        // Perform copying operation
        frontier2[idx] = frontier[idx];
    }
}