#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void executeThirdLayer(float *Layer3_Neurons_GPU, float *Layer3_Weights_GPU,float *Layer4_Neurons_GPU)
{
    int blockID = blockIdx.x;
    int weightBegin = blockID * 1251;
    float result = Layer3_Weights_GPU[weightBegin];

    // Use shared memory for Layer3_Neurons to reduce global memory access
    __shared__ float sharedLayer3_Neurons[1250];

    // Calculate a single offset for Layer3 neurons to avoid repeated calculations
    int neuronOffset = 1250 * blockIdx.y;

    if (threadIdx.x < 1250) {
        sharedLayer3_Neurons[threadIdx.x] = Layer3_Neurons_GPU[threadIdx.x + neuronOffset];
    }
    __syncthreads();

    // Use threadID to unroll the loop and parallelize computations
    for (int i = threadIdx.x; i < 1250; i += blockDim.x) {
        result += sharedLayer3_Neurons[i] * Layer3_Weights_GPU[weightBegin + i + 1];
    }

    // Use warp shuffle to reduce the results across the thread block
    __shared__ float blockResult;
    if (threadIdx.x == 0) {
        blockResult = 0;
    }
    __syncthreads();
    
    atomicAdd(&blockResult, result);
    __syncthreads();

    if (threadIdx.x == 0) {
        // Apply activation function and store result
        Layer4_Neurons_GPU[blockID + (100 * blockIdx.y)] = 1.7159 * tanhf(0.66666667 * blockResult);
    }
}
