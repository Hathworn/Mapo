#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readLocalMemory(const float *data, float *output, int size, int repeat)
{
    int gid = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    int localSize = blockDim.x;
    int grpid = blockIdx.x;
    int litems = 2048 / localSize;
    int goffset = localSize * grpid + tid * litems;
    int s = tid;
    __shared__ float lbuf[2048];
    
    // Load data into shared memory efficiently
    for (int j = 0; j < litems && j < (size - goffset); ++j)
        lbuf[tid * litems + j] = data[goffset + j];

    __syncthreads();
    
    float sum = 0.0f;
    
    // Optimize unrolling of sums
    for (int j = 0; j < repeat; ++j)
    {
        #pragma unroll 16
        for (int k = 0; k < 16; ++k)
            sum += lbuf[(s + k) & (2047)];
        
        s = (s + 16) & (2047);
    }
    
    output[gid] = sum;
}