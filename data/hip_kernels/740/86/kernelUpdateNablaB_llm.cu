#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelUpdateNablaB(float *nabla_b, float *delta_nabla_b) {
    // Use a single variable for the index to improve readability
    int idx = threadIdx.x;
    // Perform the addition using the calculated index
    nabla_b[idx] += delta_nabla_b[idx];
}