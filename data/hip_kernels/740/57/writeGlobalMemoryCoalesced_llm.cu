#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeGlobalMemoryCoalesced(float *output, int size, int repeat)
{
    int gid = threadIdx.x + (blockDim.x * blockIdx.x);
    int stride = 32768;  // Use a constant stride
    for (int j = 0; j < repeat; ++j)
    {
        #pragma unroll
        for (int i = 0; i < 16; ++i) // Unroll loop for memory access
        {
            output[(gid + i * stride) & (size - 1)] = gid;
        }
        gid = (gid + 16 * stride) & (size - 1); // Update gid for next iteration
    }
}