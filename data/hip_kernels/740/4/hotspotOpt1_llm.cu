#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hotspotOpt1(float *p, float* tIn, float *tOut, float sdc, int nx, int ny, int nz, float ce, float cw, float cn, float cs, float ct, float cb, float cc)
{
    float amb_temp = 80.0;

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    int c = i + j * nx;
    int xy = nx * ny;

    // Clamp boundary conditions to avoid conditional operator overhead
    int W = max(i - 1, 0) + j * nx;
    int E = min(i + 1, nx - 1) + j * nx;
    int N = i + max(j - 1, 0) * nx;
    int S = i + min(j + 1, ny - 1) * nx;

    float temp1, temp2, temp3;
    temp1 = temp2 = tIn[c];
    temp3 = tIn[c + xy];

    // Pre-compute terms outside the loop if possible
    float cc_temp2 = cc * temp2;
    float cb_temp1 = cb * temp1;
    float ct_amb_temp = ct * amb_temp;

    // Initial computation outside the loop
    tOut[c] = cc_temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
              + cn * tIn[N] + cb_temp1 + ct * temp3 + sdc * p[c] + ct_amb_temp;
    
    // Main loop
    c += xy;
    W += xy;
    E += xy;
    N += xy;
    S += xy;

    for (int k = 1; k < nz - 1; ++k) {
        temp1 = temp2;
        temp2 = temp3;
        temp3 = tIn[c + xy];

        cc_temp2 = cc * temp2; // Update pre-computed terms
        cb_temp1 = cb * temp1;

        tOut[c] = cc_temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
                  + cn * tIn[N] + cb_temp1 + ct * temp3 + sdc * p[c] + ct_amb_temp;
        
        c += xy;
        W += xy;
        E += xy;
        N += xy;
        S += xy;
    }
    
    // Final computation outside loop
    temp1 = temp2;
    temp2 = temp3;
    cc_temp2 = cc * temp2;
    cb_temp1 = cb * temp1;

    tOut[c] = cc_temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
              + cn * tIn[N] + cb_temp1 + ct * temp3 + sdc * p[c] + ct_amb_temp;
}