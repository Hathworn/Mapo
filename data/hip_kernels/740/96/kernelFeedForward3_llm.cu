#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelFeedForward3(float *zs, float *biases, int b_off, float *activations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index once
    float z = zs[idx] + biases[b_off + threadIdx.x]; // Use a local variable for efficiency
    activations[idx] = 1.0f / (1.0f + expf(-z)); // Directly use local variable
}