#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_one_block_spill(volatile unsigned int *frontier, unsigned int frontier_len, volatile unsigned int *cost, volatile int *visited, unsigned int *edgeArray, unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, volatile unsigned int *frontier_length, const unsigned int max_mem) {
    // Calculate thread and block indices
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory to reduce global memory access latency
    extern __shared__ unsigned int shared_frontier[];

    // Loop over the frontier in chunks of the total number of threads
    for (int i = tid; i < frontier_len; i += stride) {
        unsigned int vertex = frontier[i];
        unsigned int startEdge = edgeArray[vertex];
        unsigned int endEdge = edgeArray[vertex + 1];

        for (int edge = startEdge; edge < endEdge; ++edge) {
            unsigned int neighbor = edgeArrayAux[edge];

            // Use atomic operation to ensure each vertex is processed only once
            if (atomicExch((int *)&visited[neighbor], 1) == 0) {
                cost[neighbor] = cost[vertex] + 1;

                // Synchronize thread access to shared memory
                int index = atomicAdd((int *)frontier_length, 1);

                // Avoid memory overflow
                if (index < max_mem) {
                    shared_frontier[index] = neighbor;
                }
            }
        }
    }

    // Write back to the global memory from shared memory
    __syncthreads();
    for (int i = tid; i < *frontier_length; i += stride) {
        frontier[i] = shared_frontier[i];
    }
}