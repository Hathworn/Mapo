#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BFS_kernel_SM_block(volatile unsigned int *frontier, volatile unsigned int *frontier2, unsigned int frontier_len, volatile unsigned int *cost, volatile int *visited, unsigned int *edgeArray, unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, volatile unsigned int *frontier_length, unsigned int num_p_per_mp, unsigned int w_q_size) {
    // Calculate global thread ID for unique indexing
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Process nodes in the frontier
    if (tid < frontier_len) {
        unsigned int node = frontier[tid];

        // Access edge array to traverse adjacency list
        for (unsigned int edge_idx = edgeArrayAux[node]; edge_idx < edgeArrayAux[node + 1]; edge_idx++) {
            unsigned int neighbor = edgeArray[edge_idx]; // Get neighbor

            // Atomic operation to ensure only the first thread updates visited
            int was_visited = atomicExch(&visited[neighbor], 1);

            if (!was_visited) { // If the neighbor was not visited before
                cost[neighbor] = cost[node] + 1;  // Update cost
                unsigned int pos = atomicAdd(frontier_length, 1);  // Reserve position in frontier2
                frontier2[pos] = neighbor;  // Add new frontier node
            }
        }
    }
}