#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Block-wise kernel for better concurrent execution
__global__ void kernelFeedForward2(float *zs, float *biases, int b_off, float *activations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Unique index for each thread
    zs[idx] += biases[b_off + idx];
    activations[idx] = 1.0 / (1.0 + expf(-zs[idx]));
}