#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelBackprop3a(float *delta_nabla_b, int b_off, int bound, int b_off_old, float *weights, int w_off_old) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID

    if (tid < blockDim.x) { // Ensure we don't access out of bounds
        float temp = 0.0f; // Local variable for accumulation
        for (int j = 0; j < bound; j++) {
            // Accumulate with stride for coalesced memory access
            temp += delta_nabla_b[b_off_old + j] * weights[w_off_old + (j * blockDim.x) + tid];
        }
        delta_nabla_b[b_off + tid] = temp; // Write result back to global memory
    }
}