#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelBackprop1(float *delta_nabla_w, int w_off, float *activations, float *delta_nabla_b, int b_off) {
    // Use a single index for both block and thread for better memory coalescing
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    delta_nabla_w[w_off + index] = activations[threadIdx.x] * delta_nabla_b[b_off + blockIdx.x];
}