#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelBackprop3b(float *delta_nabla_b, int b_off, float *zs) {
    int idx = b_off + threadIdx.x; // Use variable to reduce redundant computation
    float sigmoid = 1.0f / (1.0f + expf(-zs[threadIdx.x])); // Calculate sigmoid once
    float sigmoid_derivative = sigmoid * (1.0f - sigmoid); // Use intermediate result
    delta_nabla_b[idx] *= sigmoid_derivative; // Simplify computation
}