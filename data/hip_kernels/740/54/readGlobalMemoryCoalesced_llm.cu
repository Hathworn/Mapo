#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readGlobalMemoryCoalesced(float *data, float *output, int size, int repeat)
{
    int gid = threadIdx.x + (blockDim.x * blockIdx.x), j = 0;
    float sum = 0;
    int s = gid;

    #pragma unroll // Unroll loop for better performance
    for (j = 0; j < repeat; ++j)
    {
        #pragma unroll // Unroll accesses for better coalescing and performance
        for (int i = 0; i < 16; ++i)
        {
            sum += data[(s + i * 32768) & (size - 1)];
        }
        s = (s + 524288) & (size - 1);
    }

    output[gid] = sum;
}