#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_SM_block_spill(volatile unsigned int *frontier, volatile unsigned int *frontier2, unsigned int frontier_len, volatile unsigned int *cost, volatile int *visited, unsigned int *edgeArray, unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, volatile unsigned int *frontier_length, const unsigned int max_mem) {
    // Calculate the global thread index
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;  
    
    // Iterate over all vertices in the current frontier
    for (unsigned int i = tid; i < frontier_len; i += blockDim.x * gridDim.x) {
        unsigned int vertex = frontier[i];
        // Iterate over all neighbors of the current vertex
        for (unsigned int edgeIdx = edgeArray[vertex]; edgeIdx < edgeArray[vertex + 1]; ++edgeIdx) {
            unsigned int neighbour = edgeArrayAux[edgeIdx];
            // If the neighbor has not been visited
            if (atomicExch((int*)&visited[neighbour], 1) == 0) { 
                // Update cost
                cost[neighbour] = cost[vertex] + 1;
                // Add neighbour to the new frontier
                unsigned int idx = atomicAdd(frontier_length, 1);
                frontier2[idx] = neighbour;
            }
        }
    }
}