#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_warp(unsigned int *levels, unsigned int *edgeArray, unsigned int *edgeArrayAux, int W_SZ, int CHUNK_SZ, unsigned int numVertices, int curr, int *flag) {
    // Calculate the unique warp ID within the grid
    unsigned int warpId = (blockIdx.x * blockDim.x + threadIdx.x) / W_SZ;

    // Calculate the lane index within the warp
    unsigned int laneId = threadIdx.x % W_SZ;

    // Calculate vertex for this thread to work on, stepping by the number of warps
    for (unsigned int vertex = warpId; vertex < numVertices; vertex += gridDim.x * blockDim.x / W_SZ) {
        // Perform work using warp-level primitives for efficiency
        unsigned int startEdge = edgeArray[vertex];
        unsigned int edgeCount = edgeArray[vertex + 1] - startEdge;

        // Traverse edges in chunks
        for (unsigned int edgeIndex = laneId; edgeIndex < edgeCount; edgeIndex += W_SZ) {
            unsigned int neighbor = edgeArrayAux[startEdge + edgeIndex];

            // If the level is uninitialized, set the level and update the flag
            if (levels[neighbor] == UINT_MAX) {
                levels[neighbor] = curr + 1;
                *flag = 1;
            }
        }
    }
}