#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFeedForward1b(float *zs, int bound, float *weights, int w_off, float *activations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate the global thread index

    float z_value = 0.0f;  // Initialize accumulator for the sum
    for (int i = 0; i < bound; i++) {
        z_value += weights[w_off + (threadIdx.x * bound) + i] * activations[blockIdx.x * bound + i];
    }
    zs[idx] = z_value;  // Store computed value in zs array
}