#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFeedForward1(float *zs, int bound2, float *weights, int w_off, float *activations1) {
    // Initialize z for this thread
    float z = 0.0f;

    // Use shared memory to reduce global memory accesses
    __shared__ float shared_activations1[BLOCK_SIZE];
    
    int tx = threadIdx.x;

    // Loop with stride to cover all elements
    for (int i = tx; i < bound2; i += blockDim.x) {
        if (i < bound2) { // Bounds check
            shared_activations1[i] = activations1[i];
        }
        __syncthreads(); // Synchronize after loading
                
        // Accumulate dot product
        z += weights[w_off + (tx * bound2) + i] * shared_activations1[i];
        __syncthreads();
    }

    // Write the result to global memory
    zs[tx] = z;
}