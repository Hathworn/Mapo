#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readGlobalMemoryUnit(float *data, float *output, int size, int repeat)
{
    // Pre-compute the unique global index for each thread
    int gid = threadIdx.x + (blockDim.x * blockIdx.x);
    float sum = 0;
  
    // Loop to repeat the operation as per requirement
    for (int j = 0; j < repeat; ++j)
    {
        // Iteratively load data in chunks
        #pragma unroll // Unroll loop for better performance
        for (int i = 0; i < 512; i += 16)
        {
            int base_idx = (gid * 512 + i) & (size - 1);
            sum += data[base_idx + 0] + data[base_idx + 1] + data[base_idx + 2] + data[base_idx + 3] +
                   data[base_idx + 4] + data[base_idx + 5] + data[base_idx + 6] + data[base_idx + 7] +
                   data[base_idx + 8] + data[base_idx + 9] + data[base_idx + 10]+ data[base_idx + 11] +
                   data[base_idx + 12]+ data[base_idx + 13]+ data[base_idx + 14]+ data[base_idx + 15];
        }
    }
    // Store result in global memory
    output[gid] = sum;
}