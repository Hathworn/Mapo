#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelUpdateBiases(float *nabla_b, float *biases, float eta, float mini_batch_size) {
    // Compute rate once outside the loop for efficiency
    float rate = eta / mini_batch_size;

    // Using blockIdx.x and blockDim.x for potential multi-block operations
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if index is within bounds
    if (idx < blockDim.x) {
        biases[idx] -= rate * nabla_b[idx];
    }
}