#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void executeFourthLayer(float *Layer4_Neurons_GPU, float *Layer4_Weights_GPU, float *Layer5_Neurons_GPU)
{
    int blockID = blockIdx.x * gridDim.y + blockIdx.y; // Optimize computing blockID
    int weightBegin = blockID * 101;

    float result = Layer4_Weights_GPU[weightBegin]; // Initialize result using the bias term
    ++weightBegin;

    // Using a loop unrolling optimization to enhance performance
    for (int i = 0; i < 100; i += 4)
    {
        result += Layer4_Neurons_GPU[i + (100 * blockIdx.y)] * Layer4_Weights_GPU[weightBegin + i];
        result += Layer4_Neurons_GPU[i + 1 + (100 * blockIdx.y)] * Layer4_Weights_GPU[weightBegin + i + 1];
        result += Layer4_Neurons_GPU[i + 2 + (100 * blockIdx.y)] * Layer4_Weights_GPU[weightBegin + i + 2];
        result += Layer4_Neurons_GPU[i + 3 + (100 * blockIdx.y)] * Layer4_Weights_GPU[weightBegin + i + 3];
    }

    result = (1.7159f * tanhf(0.66666667f * result)); // Apply the activation function

    Layer5_Neurons_GPU[blockID] = result; // Store the result
}