#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_one_block(volatile unsigned int *frontier, unsigned int frontier_len, volatile unsigned int *cost, volatile int *visited, unsigned int *edgeArray, unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, volatile unsigned int *frontier_length, unsigned int num_p_per_mp, unsigned int w_q_size) {
    // Calculate global thread index
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if within valid range
    if (tid < frontier_len) {
        unsigned int node = frontier[tid];
        unsigned int start_edge = edgeArray[node];
        unsigned int end_edge = (node + 1 < numVertices) ? edgeArray[node + 1] : numEdges;
        
        // Iterate over edges of the current node
        for (unsigned int edge = start_edge; edge < end_edge; ++edge) {
            unsigned int neighbor = edgeArrayAux[edge];
            
            // Check and update cost if not yet visited
            if (!visited[neighbor]) {
                cost[neighbor] = cost[node] + 1;
                visited[neighbor] = 1;
                
                // Atomic operation to update frontier length
                atomicAdd((unsigned int*)frontier_length, 1);
            }
        }
    }
}