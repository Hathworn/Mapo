#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeGlobalMemoryUnit(float *output, int size, int repeat)
{
    int gid = threadIdx.x + blockDim.x * blockIdx.x;
    int s = gid * 512;
    for (int j = 0; j < repeat; ++j)
    {
        #pragma unroll 16 // Unroll loop for better performance
        for (int i = 0; i < 16; ++i)
        {
            output[(s + i) & (size - 1)] = gid;
        }
        s = (s + 16) & (size - 1);
    }
}