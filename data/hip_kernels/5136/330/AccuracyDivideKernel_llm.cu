#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread 0 to perform division and prevent race conditions
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
    // Ensure all threads have updated data
    __syncthreads();
}