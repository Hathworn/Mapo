#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load accuracy value from global memory to a local variable
    float acc = *accuracy;

    // Divide by N
    acc /= N;

    // Store result back to global memory
    *accuracy = acc;
}