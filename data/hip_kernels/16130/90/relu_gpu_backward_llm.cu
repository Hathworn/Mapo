#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float val = (tid < N) ? indata[tid] : 0;
    ingrad[tid] = (val > 0) ? outgrad[tid] : 0;  // Use ternary operator for clarity and efficiency
}