#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use of larger stride for accessing elements to ensure better performance and load balancing
    int stride = blockDim.x * gridDim.x;
    for (int index = tid; index < N; index += stride) {
        out[index] = in[index] > 0 ? in[index] : 0; // Apply relu operation
    }
}