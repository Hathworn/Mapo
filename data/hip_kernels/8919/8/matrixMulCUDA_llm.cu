#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMulCUDA(float *C, float *A, float *B, int n)
{
    // Calculate the global row and column index for each thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    // Check boundary conditions
    if (row < n && col < n) {
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col]; // Compute partial dot-product
        }
        C[row * n + col] = sum; // Store the result
    }
}