#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMulCUDA3(float *C, float *B, float *A, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= n || col >= n) {
        return;
    }

    float sum = 0.0f;
    int rowOffset = row * n;
    
    // Loop unrolling to improve instruction level parallelism
    for (int k = 0; k < n; k += 4) {
        sum += A[rowOffset + k] * B[k * n + col];
        if (k + 1 < n) sum += A[rowOffset + k + 1] * B[(k + 1) * n + col];
        if (k + 2 < n) sum += A[rowOffset + k + 2] * B[(k + 2) * n + col];
        if (k + 3 < n) sum += A[rowOffset + k + 3] * B[(k + 3) * n + col];
    }
    C[rowOffset + col] = sum;
}