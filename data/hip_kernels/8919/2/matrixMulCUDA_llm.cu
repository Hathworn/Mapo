#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMulCUDA(float *C, float *A, float *B, int n)
{
    // Use blockIdx and blockDim to calculate global row and col
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    if (row >= n || col >= n) {
        return;
    }
    float sum = 0.0f;
    for (int k = 0; k < n; k++) {
        sum += A[row * n + k] * B[k * n + col];
    }
    C[row * n + col] = sum;
}