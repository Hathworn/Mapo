#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize the matrix multiplication kernel
__global__ void matrixMulCUDA(float *C, float *A, float *B, int n)
{
    // Calculate row and column indices for C
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Boundary check for threads
    if (row < n && col < n) {
        float sum = 0.0f;
        
        // Accumulate product sum
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        
        // Store the result in C
        C[row * n + col] = sum;
    }
}