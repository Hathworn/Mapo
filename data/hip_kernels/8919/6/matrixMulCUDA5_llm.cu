#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMulCUDA5(float *C, float *A, float *B, unsigned int n)
{
    // Define the starting and ending row and column for each thread block
    int startRow = blockIdx.y * blockDim.y + threadIdx.y;
    int startCol = blockIdx.x * blockDim.x + threadIdx.x;

    // Load matrices dimensions
    const int tileWidth = blockDim.x;
    const int tileHeight = blockDim.y;

    // Allocate shared memory for A and B tiles
    __shared__ float A_S[32 * 32];
    __shared__ float B_S[32 * 32];

    // Initialize compute register
    float sum = 0.0f;

    // Loop over tiles
    for (int tileIdx = 0; tileIdx < n / tileWidth; ++tileIdx) {
        // Load tile from global memory into shared memory
        int AstartIdx = startRow * n + tileIdx * tileWidth + threadIdx.x;
        int BstartIdx = (tileIdx * tileHeight + threadIdx.y) * n + startCol;

        A_S[threadIdx.y * tileWidth + threadIdx.x] = A[AstartIdx];
        B_S[threadIdx.y * tileWidth + threadIdx.x] = B[BstartIdx];

        // Synchronize all threads to make sure the tile is loaded
        __syncthreads();

        // Compute partial result
        for (int k = 0; k < tileWidth; ++k) {
            sum += A_S[threadIdx.y * tileWidth + k] * B_S[k * tileWidth + threadIdx.x];  
        }

        // Synchronize again to avoid any overlap
        __syncthreads();
    }

    // Write final result to global memory
    C[startRow * n + startCol] = sum;
}