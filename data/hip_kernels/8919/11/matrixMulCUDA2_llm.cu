#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define TILE_WIDTH 16

__global__ void matrixMulCUDA2(float *C, float *A, float *B, int n)
{
    // Calculate row and column index for each thread
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    // Initialize sum for the current element (register optimization)
    float sum = 0.0f;

    // Ensure threads within bounds
    if (row < n && col < n) {
        // Compute dot product for the row and column
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }

        // Write back the computed value to global memory
        C[row * n + col] = sum;
    }
}