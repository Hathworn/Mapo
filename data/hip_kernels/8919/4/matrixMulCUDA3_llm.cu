#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMulCUDA3(float *C, float *B, float *A, int n)
{
    // Calculate the row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize sum variable
    float sum = 0.0f;

    // Check if row and col are within matrix size
    if (row < n && col < n) {
        // Loop to perform dot product for the matrix multiplication
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        // Store the result
        C[row * n + col] = sum;
    }
}