#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__global__ void matrixMulCUDA2(float *C, float *A, float *B, int n)
{
    // Compute global row and column indexes
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds
    if (row < n && col < n) {
        // Calculate the dot product for C[row, col]
        float sum = 0.0f;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        // Store the result in C
        C[row * n + col] = sum;
    }
}