#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// GPU Libraries

// Macro to handle errors occurred in CUDA api
__device__ void recursiveReduce(int *g_inData, int *g_outData, int inSize, int outSize)
{
    extern __shared__ int sData[];

    // Identification
    unsigned int tId = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize
    sData[tId] = 0;

    __syncthreads();

    // Fill up the shared memory
    if (i < inSize) { // Ensure within bounds 
        sData[tId] = g_inData[i];
    }

    __syncthreads();

    // Tree based reduction
    for (unsigned int d = blockDim.x / 2; d > 0; d /= 2) { // Use half-interval step 
        if (tId < d) // Only necessary threads
            sData[tId] += sData[tId + d];

        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tId == 0) {
        g_outData[blockIdx.x] = sData[0];
    }

    // Recursive call
    if (outSize > 1 && blockIdx.x == 0 && tId == 0) { // Ensure only one recursive call
        // Kernel Launch
        recursiveReduce(g_outData, g_outData, outSize, (outSize - 1) / blockDim.x + 1);
    }
}

__global__ void reduceKernel(int *g_inData, int *g_outData, int inSize, int outSize)
{
    recursiveReduce(g_inData, g_outData, inSize, outSize);
}