#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMulCUDA4(float *C, float *A, float *B, unsigned int n)
{
    // Obtain block index and thread index within the block
    int blockRow = blockIdx.y * blockDim.y;
    int blockCol = blockIdx.x * blockDim.x;
    int threadRow = threadIdx.y;
    int threadCol = threadIdx.x;

    // Compute global row and column indices
    int row = blockRow + threadRow;
    int col = blockCol + threadCol;

    // Check boundary conditions
    if (row < n && col < n) {
        // Compute sum using shared memory
        float sum = 0.0f;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }

        // Write the result to the output matrix
        C[row * n + col] = sum;
    }
}