#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int g_tId = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate warpId without redundant operation
    unsigned int warpId = g_tId / warpSize;

    if (g_tId < vectorSize) {
        c[g_tId] = a[g_tId] + b[g_tId];

        // Use conditional compilation to remove printf in release builds
        #ifdef DEBUG
        printf("thread id %d , warp Id %d , block id %d\n", g_tId, warpId, blockIdx.x);
        #endif
    }
}