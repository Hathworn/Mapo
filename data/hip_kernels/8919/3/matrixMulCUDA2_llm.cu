#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMulCUDA2(float *C, float *A, float *B, int n)
{
    // Utilize shared memory to reduce global memory accesses
    __shared__ float sharedA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float sharedB[TILE_WIDTH][TILE_WIDTH];

    // Calculate indices for the global memory
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    float sum = 0.0f;

    // Loop over tiles of the input matrices
    for (int tile = 0; tile < (n + TILE_WIDTH - 1) / TILE_WIDTH; ++tile) {
        
        // Load data into shared memory, checking bounds
        if (row < n && tile * TILE_WIDTH + threadIdx.x < n)
            sharedA[threadIdx.y][threadIdx.x] = A[row * n + tile * TILE_WIDTH + threadIdx.x];
        else
            sharedA[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < n && tile * TILE_WIDTH + threadIdx.y < n)
            sharedB[threadIdx.y][threadIdx.x] = B[(tile * TILE_WIDTH + threadIdx.y) * n + col];
        else
            sharedB[threadIdx.y][threadIdx.x] = 0.0f;

        // Synchronize to make sure the tile is loaded
        __syncthreads();

        // Perform the computation for the tile
        for (int k = 0; k < TILE_WIDTH; ++k) {
            sum += sharedA[threadIdx.y][k] * sharedB[k][threadIdx.x];
        }

        // Synchronize to ensure all computations using shared memory are done
        __syncthreads();
    }

    // Store the result in global memory
    if (row < n && col < n) {
        C[row * n + col] = sum;
    }
}