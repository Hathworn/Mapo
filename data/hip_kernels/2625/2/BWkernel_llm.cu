#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

typedef struct bmpFileHeaderStruct {
    uint32_t size;
    uint16_t resv1;
    uint16_t resv2;
    uint32_t offset;
} bmpFileHeader;

typedef struct bmpInfoHeaderStruct {
    uint32_t headersize;
    uint32_t width;
    uint32_t height;
    uint16_t planes;
    uint16_t bpp;
    uint32_t compress;
    uint32_t imgsize;
    uint32_t bpmx;
    uint32_t bpmy;
    uint32_t colors;
    uint32_t imxtcolors;
} bmpInfoHeader;


__global__ void BWkernel(unsigned char *img_device, uint32_t n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        // Perform the conversion in one step to save computation time
        unsigned char r = img_device[i * 3 + 0];
        unsigned char g = img_device[i * 3 + 1];
        unsigned char b = img_device[i * 3 + 2];
        unsigned char color = static_cast<unsigned char>(0.299 * b + 0.587 * g + 0.114 * r);
        
        // Directly assign the converted grayscale value to RGB components
        img_device[i * 3 + 0] = color;
        img_device[i * 3 + 1] = color;
        img_device[i * 3 + 2] = color;
    }
}