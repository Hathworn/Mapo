#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

typedef struct bmpFileHeaderStruct {
/* 2 bytes de identificación */
uint32_t size;        /* Tamaño del archivo */
uint16_t resv1;       /* Reservado */
uint16_t resv2;       /* Reservado */
uint32_t offset;      /* Offset hasta hasta los datos de imagen */
} bmpFileHeader;

typedef struct bmpInfoHeaderStruct {
uint32_t headersize;  /* Tamaño de la cabecera */
uint32_t width;       /* Ancho */
uint32_t height;      /* Alto */
uint16_t planes;      /* Planos de color (Siempre 1) */
uint16_t bpp;         /* bits por pixel */
uint32_t compress;    /* compresion */
uint32_t imgsize;     /* tamaño de los datos de imagen */
uint32_t bpmx;        /* Resolucion X en bits por metro */
uint32_t bpmy;        /* Resolucion Y en bits por metro */
uint32_t colors;      /* colors used en la paleta */
uint32_t imxtcolors;  /* Colores importantes. 0 si son todos */
} bmpInfoHeader;

__global__ void blurKernel(unsigned char *img_device, unsigned char *img_device2, uint32_t width_image, uint32_t height_image) {

    int x, y;
    int blurSize = 10;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int i = width_image * row + col;

    if (row < height_image && col < width_image) { // Check bounds for threads
        int avgR = 0, avgG = 0, avgB = 0, ile = 0;

        // Optimize loop bounds calculation
        int startX = max(0, col - blurSize / 2);
        int startY = max(0, row - blurSize / 2);
        int endX = min(width_image, col + blurSize / 2 + 1);
        int endY = min(height_image, row + blurSize / 2 + 1);

        for (x = startX; x < endX; x++) {
            for (y = startY; y < endY; y++) {
                int idx = (x * 3) + (y * width_image * 3);
                avgB += img_device2[idx + 0];
                avgG += img_device2[idx + 1];
                avgR += img_device2[idx + 2];
                ile++;
            }
        }

        avgB /= ile;
        avgG /= ile;
        avgR /= ile;

        int outIdx = (col * 3) + (row * width_image * 3);
        img_device[outIdx + 0] = avgB;
        img_device[outIdx + 1] = avgG;
        img_device[outIdx + 2] = avgR;
    }
}