#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

typedef struct bmpFileHeaderStruct {
    uint32_t size;      
    uint16_t resv1;     
    uint16_t resv2;     
    uint32_t offset;    
} bmpFileHeader;

typedef struct bmpInfoHeaderStruct {
    uint32_t headersize;
    uint32_t width;     
    uint32_t height;    
    uint16_t planes;    
    uint16_t bpp;       
    uint32_t compress;  
    uint32_t imgsize;   
    uint32_t bpmx;      
    uint32_t bpmy;      
    uint32_t colors;    
    uint32_t imxtcolors;
} bmpInfoHeader;

__global__ void ConvMatKernel(unsigned char *img_device, unsigned char *img_device2, uint32_t width_image, uint32_t height_image, int j, float *mat) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread is within image bounds
    if (row >= height_image || col >= width_image) return;

    float avgB = 0.0f, avgG = 0.0f, avgR = 0.0f;

    // Apply convolution only if pixel is within bounds
    for (int x = -1; x <= 1; x++) {
        for (int y = -1; y <= 1; y++) {
            int newRow = row + x;
            int newCol = col + y;
            if (newRow >= 0 && newRow < height_image && newCol >= 0 && newCol < width_image) {
                avgB += img_device[(newCol*3) + (newRow*width_image*3) + 0] * mat[(x + 1) * 3 + (y + 1)];
                avgG += img_device[(newCol*3) + (newRow*width_image*3) + 1] * mat[(x + 1) * 3 + (y + 1)];
                avgR += img_device[(newCol*3) + (newRow*width_image*3) + 2] * mat[(x + 1) * 3 + (y + 1)];
            }
        }
    }

    int imgIndex = col*3 + row*width_image*3;
    img_device2[imgIndex + 0] = avgB;
    img_device2[imgIndex + 1] = avgG;
    img_device2[imgIndex + 2] = avgR;
}