#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < size) // Ensure thread does not exceed array bounds
    {
        c[i] = a[i] + b[i];
    }
}