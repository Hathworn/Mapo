#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation
    /* Initialize each thread with the same seed, different sequence number, no offset */
    hiprand_init(seed, tidx, 0, &state[tidx]);
}