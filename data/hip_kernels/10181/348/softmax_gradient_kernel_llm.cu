#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    // Perform parallel reduction to calculate the inner product with fewer steps
    float tmp = 0;
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Reduce with a single warp loop for better performance when blockDim.x > warpSize
    if (idx < warpSize) {
        for (int i = idx + warpSize; i < blockDim.x; i += warpSize) {
            tmp += reduction_buffer[i];
        }
        reduction_buffer[idx] = tmp;
        __syncthreads();

        if (idx == 0) {
            tmp = 0;
            for (int i = 0; i < warpSize; ++i) {
                tmp += reduction_buffer[i];
            }
            reduction_buffer[0] = tmp;
        }
    }
    __syncthreads();

    // Compute gradient
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}