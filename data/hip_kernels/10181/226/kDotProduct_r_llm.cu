#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;

    // Unrolling the loop for better performance
    if (eidx < numElements) {
        unsigned int stride = gridDim.x * DP_BLOCKSIZE;
        while (eidx < numElements) {
            shmem[threadIdx.x] += a[eidx] * b[eidx];
            eidx += stride;
        }
    }

    __syncthreads();

    // Reduce the requirement for many __syncthreads() by using warp shuffle instructions
    if (threadIdx.x < 32) {
        for (int offset = 32; offset > 0; offset /= 2) {
            shmem[threadIdx.x] += __shfl_down_sync(0xffffffff, shmem[threadIdx.x], offset);
        }

        // Final result is written to global memory by the first thread
        if (threadIdx.x == 0) {
            target[blockIdx.x] = shmem[0];
        }
    }
}