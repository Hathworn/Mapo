#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Using atomic operation for potential concurrent access
    atomicExch(accuracy, *accuracy / N);
}