#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate a unique index for each thread
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Total number of threads in the grid
    const int numThreads = blockDim.x * gridDim.x;

    // Pre-calculate total number of elements
    const unsigned int numEls = tgtWidth * tgtHeight;

    // Loop through target elements each thread should handle
    for (uint i = idx; i < numEls; i += numThreads) {
        // Calculate y and x based on index
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;

        // Calculate source y and x with modulus operator
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Efficient memory access by calculating source index once
        const uint srcIdx = srcY * srcWidth + srcX;

        // Assign value from source to target
        tgt[i] = src[srcIdx];
    }
}