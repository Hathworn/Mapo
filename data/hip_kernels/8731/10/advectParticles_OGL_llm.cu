#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void advectParticles_OGL(float2 *part, float2 *v, int dx, int dy, float dt, int lb, size_t pitch) {
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;

    // Exit early if the thread is out of bounds in x-dimension, added for clarity
    if (gtidx >= dx) return;

    for (int p = 0; p < lb; p++) {
        int fi = gtidy + p;

        // Exit early if the thread is out of bounds in y-dimension
        if (fi >= dy) return;

        int fj = fi * dx + gtidx;
        float2 pterm = part[fj];

        int xvi = min(max((int)(pterm.x * dx), 0), dx - 1); // Ensure xvi is within bounds
        int yvi = min(max((int)(pterm.y * dy), 0), dy - 1); // Ensure yvi is within bounds
        float2 vterm = *((float2*)((char*)v + yvi * pitch) + xvi);

        pterm.x = fmodf(pterm.x + dt * vterm.x, 1.f); // Optimized wrap around for pterm.x
        if (pterm.x < 0.f) pterm.x += 1.f; // Ensures positive modulus result

        pterm.y = fmodf(pterm.y + dt * vterm.y, 1.f); // Optimized wrap around for pterm.y
        if (pterm.y < 0.f) pterm.y += 1.f; // Ensures positive modulus result

        part[fj] = pterm;
    }
}