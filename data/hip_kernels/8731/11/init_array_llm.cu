#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
    // Cache factor value in shared memory for faster access
    __shared__ int cached_factor;
    if (threadIdx.x == 0) {
        cached_factor = *factor;
    }
    __syncthreads();

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use local variable for g_data update to avoid redundant memory access
    int local_value = g_data[idx];
    for (int i = 0; i < num_iterations; i++)
        local_value += cached_factor;
    g_data[idx] = local_value; // Write result back to global memory
}