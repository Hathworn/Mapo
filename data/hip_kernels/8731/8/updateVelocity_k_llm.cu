#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateVelocity_k(float2 *v, float *vx, float *vy, int dx, int pdx, int dy, int lb, size_t pitch) {
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;

    // Pre-calculating scale factor to avoid repetitive division
    float scale = 1.f / (dx * dy);

    if (gtidx < dx) {
        for (int p = 0; p < lb; p++) {
            int fi = gtidy + p;
            if (fi < dy) {
                // Reduced number of array index calculations
                int fjr = fi * pdx + gtidx;
                
                float vxterm = vx[fjr];
                float vyterm = vy[fjr];

                float2 nvterm;
                nvterm.x = vxterm * scale;
                nvterm.y = vyterm * scale;

                // Simplified pointer arithmetic for memory access
                float2 *fj = (float2*)((char*)v + fi * pitch) + gtidx;
                *fj = nvterm;
            }
        }
    }
}