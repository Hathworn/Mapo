#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diffuseProject_k(float2 *vx, float2 *vy, int dx, int dy, float dt, float visc, int lb) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * blockDim.y + threadIdx.y; // Optimize for coalesced memory access

    if (gtidx < dx) {
        for (int p = 0; p < lb; p++) {
            int fi = gtidy * lb + p; // Align loop index for better cache usage
            if (fi < dy) {
                int fj = fi * dx + gtidx;
                float2 xterm = vx[fj];
                float2 yterm = vy[fj];

                // Compute the index of the wavenumber based on the
                // data order produced by a standard NN FFT.
                int iix = gtidx;
                int iiy = (fi > dy / 2) ? (fi - dy) : fi;

                // Velocity diffusion
                float kk = (float)(iix * iix + iiy * iiy); // k^2
                float diff = 1.f / (1.f + visc * dt * kk);
                xterm.x *= diff; xterm.y *= diff;
                yterm.x *= diff; yterm.y *= diff;

                // Velocity projection
                if (kk > 0.f) {
                    float rkk = 1.f / kk;
                    float rkp = (iix * xterm.x + iiy * yterm.x);
                    float ikp = (iix * xterm.y + iiy * yterm.y);
                    xterm.x -= rkk * rkp * iix;
                    xterm.y -= rkk * ikp * iix;
                    yterm.x -= rkk * rkp * iiy;
                    yterm.y -= rkk * ikp * iiy;
                }

                vx[fj] = xterm;
                vy[fj] = yterm;
            }
        }
    }
}