#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function for adding forces
__global__ void addForces_k(float2 *v, int dx, int dy, int spx, int spy, float fx, float fy, int r, size_t pitch) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float2 *fj = (float2*)((char*)v + (ty + spy) * pitch) + (tx + spx);

    // Pre-calculate the repositioned indices
    int x = tx - r;
    int y = ty - r;
    
    // Optimize the computation of s using a single multiplication instead of multiple divisions
    float invDenom = 1.f / (1.f + x*x*x*x + y*y*y*y);
    
    // Merge arithmetic operations for efficiency
    fj->x += invDenom * fx;
    fj->y += invDenom * fy;
}