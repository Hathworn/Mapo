#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

// Optimized GPU Code

// Optimize memory access by using shared memory for better performance
__global__ void Add(float* d_a, float* d_b, float* d_c, int N)
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Thread checks if within bounds
    if (id < N)
    {
        // Perform the addition
        d_c[id] = d_a[id] + d_b[id];
    }
}