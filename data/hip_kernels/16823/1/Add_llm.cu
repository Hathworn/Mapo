#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Add(float* d_a, float* d_b, float* d_c, int N)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < N)
    {
        // Prevents redundant loads by using registers
        float a_val = d_a[id];
        float b_val = d_b[id];
        d_c[id] = a_val + b_val;
    }
}