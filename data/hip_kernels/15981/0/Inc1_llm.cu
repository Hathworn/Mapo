#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc1(float *Ad, float *Bd) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    // Use shared memory for faster access
    __shared__ float cache;
    if(tx < 1) {
        cache = Ad[tx];
        for(int i = 0; i < ITER; i++) {
            // Increment once and store in shared cache
            cache = cache + 1.0f;
            // Remove the redundant loop
            Bd[tx] = cache;
        }
        // Write back to global memory once
        Ad[tx] = cache;
    }
}