#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc2(float *Ad, float *Bd) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure tx is within bounds
    if(tx < LEN){
        // Cache Ad[tx] to reduce global memory access
        float aVal = Ad[tx];
        for(int i = 0; i < ITER; i++) {
            aVal += 1.0f; // Increment cached value
            // Use shared variable to avoid redundant writes
            float bVal = aVal;
            for(int j = 0; j < 256; j++) {
                Bd[tx] = bVal;
            }
        }
        // Write back to global memory after loop
        Ad[tx] = aVal;
    }
}