#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define UPPERTHRESHOLD 90
#define LOWERTHRESHOLD 30

const float G_x[3 * 3] = {
-1, 0, 1,
-2, 0, 2,
-1, 0, 1
};

const float G_y[3 * 3] = {
1, 2, 1,
0, 0, 0,
-1, -2, -1
};

const float gaussian[5 * 5] = {
2.f/159, 4.f/159, 5.f/159, 4.f/159, 2.f/159,
4.f/159, 9.f/159, 12.f/159, 9.f/159, 4.f/159,
5.f/159, 12.f/159, 15.f/159, 12.f/159, 2.f/159,
4.f/159, 9.f/159, 12.f/159, 9.f/159, 4.f/159,
2.f/159, 4.f/159, 5.f/159, 4.f/159, 2.f/159
};

__global__ void kernGradient(int N, int width, int height, unsigned char * in, unsigned char * gradient, unsigned char * edgeDir, float * G_x, float * G_y) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (x >= width || y >= height) return;
    
    int idx = y * width + x;
    float Gx = 0.0f, Gy = 0.0f;
    
    // Use shared memory to reduce global memory access latency
    __shared__ unsigned char shared_in[32][32];
    int local_x = threadIdx.x + 1;
    int local_y = threadIdx.y + 1;
    
    shared_in[local_y][local_x] = in[idx];
    if (threadIdx.x == 0 && x > 0) {
        shared_in[local_y][0] = in[idx - 1];
    }
    if (threadIdx.x == blockDim.x - 1 && x < width - 1) {
        shared_in[local_y][blockDim.x + 1] = in[idx + 1];
    }
    if (threadIdx.y == 0 && y > 0) {
        shared_in[0][local_x] = in[idx - width];
    }
    if (threadIdx.y == blockDim.y - 1 && y < height - 1) {
        shared_in[blockDim.y + 1][local_x] = in[idx + width];
    }
    __syncthreads();
    
    // Compute Gx and Gy using shared memory
    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            Gx += shared_in[local_y + dy][local_x + dx] * G_x[(dy + 1) * 3 + (dx + 1)];
            Gy += shared_in[local_y + dy][local_x + dx] * G_y[(dy + 1) * 3 + (dx + 1)];
        }
    }
    
    float grad = sqrtf(Gx * Gx + Gy * Gy);
    float angle = atan2f(Gy, Gx) * (180.0f / 3.14159f);
    
    unsigned char roundedAngle;
    if (((-22.5 < angle) && (angle <= 22.5)) || ((157.5 < angle) || (angle <= -157.5))) {
        roundedAngle = 0;
    } else if (((-157.5 < angle) && (angle <= -112.5)) || ((22.5 < angle) && (angle <= 67.5))) {
        roundedAngle = 45;
    } else if (((-112.5 < angle) && (angle <= -67.5)) || ((67.5 < angle) && (angle <= 112.5))) {
        roundedAngle = 90;
    } else {
        roundedAngle = 135;
    }
    
    gradient[idx] = grad;
    edgeDir[idx] = roundedAngle;
}