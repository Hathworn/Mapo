#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define UPPERTHRESHOLD 90
#define LOWERTHRESHOLD 30

const float G_x[3 * 3] = {
-1, 0, 1,
-2, 0, 2,
-1, 0, 1
};

const float G_y[3 * 3] = {
1, 2, 1,
0, 0, 0,
-1, -2, -1
};

const float gaussian[5 * 5] = {
2.f/159, 4.f/159, 5.f/159, 4.f/159, 2.f/159,
4.f/159, 9.f/159, 12.f/159, 9.f/159, 4.f/159,
5.f/159, 12.f/159, 15.f/159, 12.f/159, 2.f/159,
4.f/159, 9.f/159, 12.f/159, 9.f/159, 4.f/159,
2.f/159, 4.f/159, 5.f/159, 4.f/159, 2.f/159
};

__global__ void hysteresis(int N, int width, int height, unsigned char *in) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) {
        return; // Return if out of bounds
    }
    int idx = y * width + x;
    if (in[idx] > UPPERTHRESHOLD) {
        in[idx] = 255;
        return; // Directly return if above threshold
    } else if (in[idx] < LOWERTHRESHOLD) {
        in[idx] = 0;
        return; // Directly return if below threshold
    }
    // Use shared memory for neighbor pixel values
    __shared__ unsigned char local[3][3];
    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            int nx = x + dx;
            int ny = y + dy;
            if (0 <= nx && nx < width && 0 <= ny && ny < height) {
                local[dy+1][dx+1] = in[ny * width + nx]; // Load into shared memory
            } else {
                local[dy+1][dx+1] = 0; // Boundary condition
            }
        }
    }
    __syncthreads(); // Ensure all threads have loaded their pixel values
    
    // Check neighbors in shared memory
    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            if (local[dy+1][dx+1] > LOWERTHRESHOLD) {
                in[idx] = 255;
                return; // Break out once a valid neighbor is found
            }
        }
    }
}