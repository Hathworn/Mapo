#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define UPPERTHRESHOLD 90
#define LOWERTHRESHOLD 30

const float G_x[3 * 3] = {
    -1, 0, 1,
    -2, 0, 2,
    -1, 0, 1
};

const float G_y[3 * 3] = {
    1, 2, 1,
    0, 0, 0,
    -1, -2, -1
};

const float gaussian[5 * 5] = {
    2.f/159, 4.f/159, 5.f/159, 4.f/159, 2.f/159,
    4.f/159, 9.f/159, 12.f/159, 9.f/159, 4.f/159,
    5.f/159, 12.f/159, 15.f/159, 12.f/159, 2.f/159,
    4.f/159, 9.f/159, 12.f/159, 9.f/159, 4.f/159,
    2.f/159, 4.f/159, 5.f/159, 4.f/159, 2.f/159
};

__global__ void kernSmooth(int N, int width, int height, unsigned char *in, unsigned char *out, const float *kernel, int kernSize) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (x >= width || y >= height) return;

    // Initialize accumulation variable
    float c = 0.0f;
    int halfKern = kernSize / 2;

    // Iterate over kernel
    for (int i = -halfKern; i <= halfKern; i++) {
        int tx = x + i;
        if (tx >= 0 && tx < width) {
            for (int j = -halfKern; j <= halfKern; j++) {
                int ty = y + j;
                if (ty >= 0 && ty < height) {
                    // Accumulate weighted sum
                    c += in[ty * width + tx] * kernel[(j + halfKern) * kernSize + (i + halfKern)];
                }
            }
        }
    }

    // Store the absolute value of the result
    out[y * width + x] = fabs(c);
}