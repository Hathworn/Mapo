#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernGaussianBlur(int width, int height, uint8_t * dst, uint8_t * src) {
    // Calculate pixel's location in the source image
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    // Exit if the thread is out of the image boundaries
    if (x >= width || y >= height) {
        return;
    }
    
    // Define Gaussian kernel
    __shared__ float kernel[5][5];
    kernel[0][0] = 0.003765; kernel[0][1] = 0.015019; kernel[0][2] = 0.023792; kernel[0][3] = 0.015019; kernel[0][4] = 0.003765;
    kernel[1][0] = 0.015019; kernel[1][1] = 0.059912; kernel[1][2] = 0.094907; kernel[1][3] = 0.059912; kernel[1][4] = 0.015019;
    kernel[2][0] = 0.023792; kernel[2][1] = 0.094907; kernel[2][2] = 0.150342; kernel[2][3] = 0.094907; kernel[2][4] = 0.023792;
    kernel[3][0] = 0.015019; kernel[3][1] = 0.059912; kernel[3][2] = 0.094907; kernel[3][3] = 0.059912; kernel[3][4] = 0.015019;
    kernel[4][0] = 0.003765; kernel[4][1] = 0.015019; kernel[4][2] = 0.023792; kernel[4][3] = 0.015019; kernel[4][4] = 0.003765;
    
    // Initialize RGB values
    float r = 0.0f, g = 0.0f, b = 0.0f;
    
    // Compute Gaussian blur
    for (int i = 0; i < 5; i++) {
        int tx = x + i - 2;
        for (int j = 0; j < 5; j++) {
            int ty = y + j - 2;
            if (tx >= 0 && ty >= 0 && tx < width && ty < height) {
                int srcIdx = (ty * width + tx) * 3;
                r += src[srcIdx] * kernel[i][j];
                g += src[srcIdx + 1] * kernel[i][j];
                b += src[srcIdx + 2] * kernel[i][j];
            }
        }
    }
    
    // Assign computed color to destination image
    int idx = 3 * (y * width + x);
    dst[idx] = __saturatef(r);
    dst[idx + 1] = __saturatef(g);
    dst[idx + 2] = __saturatef(b);
}