#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernGaussianBlur(int width, int height, uint8_t * dst, uint8_t * src) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (x >= width || y >= height) {
        return;
    }

    // Pre-calculated constant kernel to avoid repetitive array initialization
    __shared__ float kernel[5][5];
    if (threadIdx.x < 5 && threadIdx.y < 5) {
        kernel[threadIdx.y][threadIdx.x] = 0;
        __syncthreads();
        if (threadIdx.y == 0) {
            kernel[0][0] = 0.003765f; kernel[0][1] = 0.015019f; kernel[0][2] = 0.023792f;
            kernel[0][3] = 0.015019f; kernel[0][4] = 0.003765f;
        }
        if (threadIdx.y == 1) {
            kernel[1][0] = 0.015019f; kernel[1][1] = 0.059912f; kernel[1][2] = 0.094907f;
            kernel[1][3] = 0.059912f; kernel[1][4] = 0.015019f;
        }
        if (threadIdx.y == 2) {
            kernel[2][0] = 0.023792f; kernel[2][1] = 0.094907f; kernel[2][2] = 0.150342f;
            kernel[2][3] = 0.094907f; kernel[2][4] = 0.023792f;
        }
        if (threadIdx.y == 3) {
            kernel[3][0] = 0.015019f; kernel[3][1] = 0.059912f; kernel[3][2] = 0.094907f;
            kernel[3][3] = 0.059912f; kernel[3][4] = 0.015019f;
        }
        if (threadIdx.y == 4) {
            kernel[4][0] = 0.003765f; kernel[4][1] = 0.015019f; kernel[4][2] = 0.023792f;
            kernel[4][3] = 0.015019f; kernel[4][4] = 0.003765f;
        }
    }
    __syncthreads();

    float r = 0.0f, g = 0.0f, b = 0.0f;
    // Loop optimization: combining boundary check and computation
    for (int i = 0; i < 5; ++i) {
        int tx = x + i - 2;
        if (tx >= 0 && tx < width) {
            for (int j = 0; j < 5; ++j) {
                int ty = y + j - 2;
                if (ty >= 0 && ty < height) {
                    int index = (ty * width + tx) * 3;
                    r += src[index] * kernel[i][j];
                    g += src[index + 1] * kernel[i][j];
                    b += src[index + 2] * kernel[i][j];
                }
            }
        }
    }
    int idx = 3 * (y * width + x);
    dst[idx] = (uint8_t)r;
    dst[idx + 1] = (uint8_t)g;
    dst[idx + 2] = (uint8_t)b;
    return;
}