#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define UPPERTHRESHOLD 90
#define LOWERTHRESHOLD 30

const float G_x[3 * 3] = {
-1, 0, 1,
-2, 0, 2,
-1, 0, 1
};

const float G_y[3 * 3] = {
1, 2, 1,
0, 0, 0,
-1, -2, -1
};

const float gaussian[5 * 5] = {
2.f/159, 4.f/159, 5.f/159, 4.f/159, 2.f/159,
4.f/159, 9.f/159, 12.f/159, 9.f/159, 4.f/159,
5.f/159, 12.f/159, 15.f/159, 12.f/159, 2.f/159,
4.f/159, 9.f/159, 12.f/159, 9.f/159, 4.f/159,
2.f/159, 4.f/159, 5.f/159, 4.f/159, 2.f/159
};

__global__ void nonMaxSuppression(int N, int width, int height, unsigned char * in, unsigned char * out) {
    int D = 1;
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    // Ensure the thread is within valid pixel range
    if (x >= width || y >= height) return;

    int idx = y * width + x;
    int angle = in[idx];
    unsigned char currentPixel = out[idx];

    // Perform non-maximum suppression based on the angle
    if (angle == 0) {
        if (currentPixel < out[(y + D) * width + x] || currentPixel < out[(y - D) * width + x]) {
            out[idx] = 0;
        }
    } else if (angle == 45) {
        if (currentPixel < out[(y + D) * width + x - D] || currentPixel < out[(y - D) * width + x + D]) {
            out[idx] = 0;
        }
    } else if (angle == 90) {
        if (currentPixel < out[y * width + (x + D)] || currentPixel < out[y * width + (x - D)]) {
            out[idx] = 0;
        }
    } else if (angle == 135) {
        if (currentPixel < out[(y + D) * width + x + D] || currentPixel < out[(y - D) * width + x - D]) {
            out[idx] = 0;
        }
    }
}