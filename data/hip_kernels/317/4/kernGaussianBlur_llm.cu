#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernGaussianBlur(int width, int height, uint8_t* dst, uint8_t* src, int kernSize, float* kernel) {
    // Calculate absolute position in the image
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Early exit for out-of-bounds threads
    if (x >= width || y >= height) return;

    float r = 0.0f, g = 0.0f, b = 0.0f;

    // Loop over the kernel
    for (int i = 0; i < kernSize; i++) {
        int tx = x + i - kernSize / 2;
        for (int j = 0; j < kernSize; j++) {
            int ty = y + j - kernSize / 2;
            if (tx >= 0 && ty >= 0 && tx < width && ty < height) {
                int srcIdx = (ty * width + tx) * 3;
                float kernelVal = kernel[j * kernSize + i];
                r += src[srcIdx] * kernelVal;
                g += src[srcIdx + 1] * kernelVal;
                b += src[srcIdx + 2] * kernelVal;
            }
        }
    }

    // Calculate output index and assign values
    int idx = 3 * (y * width + x);
    dst[idx] = r;
    dst[idx + 1] = g;
    dst[idx + 2] = b;
}