#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernCopyToFrame(int N, uint8_t * frame, float * src) {
    // Calculate global thread index
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    // Use a single return statement
    if (idx < N) {
        frame[idx] = src[idx] < 0 ? 0 : (uint8_t) src[idx];
    }
}