#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index
    if (i < N) // Check to avoid out-of-bounds access
    {
        c[i] = a[i] + b[i];
    }
}
```
