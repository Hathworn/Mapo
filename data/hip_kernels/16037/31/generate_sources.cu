#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generate_sources(hiprandState *state, int n, uint32_t *verts) {
int first = threadIdx.x + blockIdx.x * blockDim.x;
int stride = blockDim.x * gridDim.x;

hiprandState local_state = state[first];
for (int id = first ; id < n ; id += stride) {
verts[id] = hiprand(&local_state);
}

state[first] = local_state;
}