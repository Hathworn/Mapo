#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_sources(hiprandState *state, int n, uint32_t *verts) {
    int first = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Initialize local random state
    hiprandState local_state = state[first];

    // Unroll loop for better performance
    for (int id = first; id < n; id += stride * 4) {
        if (id < n) {
            verts[id] = hiprand(&local_state);
        }
        if (id + stride < n) {
            verts[id + stride] = hiprand(&local_state);
        }
        if (id + 2 * stride < n) {
            verts[id + 2 * stride] = hiprand(&local_state);
        }
        if (id + 3 * stride < n) {
            verts[id + 3 * stride] = hiprand(&local_state);
        }
    }

    // Store local random state back to global state
    state[first] = local_state;
}