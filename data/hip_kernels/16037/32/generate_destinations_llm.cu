#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_destinations(hiprandState *state, int n, const uint32_t *sources, uint32_t *destinations) {
    int first = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Load state into a local variable for increased performance
    hiprandState local_state = state[first];

    // Process each element using the thread's stride for better memory access
    for (int id = first; id < n; id += stride) {
        // Generate a random index and access `sources` array
        destinations[id] = sources[hiprand(&local_state) % n];
    }

    // Store the updated state back to global memory
    state[first] = local_state;
}