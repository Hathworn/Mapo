#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void myset(unsigned long long *p, unsigned long long v, long long n) {
    // Calculate global thread index
    long long tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index is within bounds
    if (tid < n) {
        p[tid] = v;  // Set value
    }
}