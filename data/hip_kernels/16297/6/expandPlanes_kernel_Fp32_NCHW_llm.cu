#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expandPlanes_kernel_Fp32_NCHW(float* output, const uint64_t* masks, const float* values, int n) {
    // Use shared memory to reduce global memory accesses, improving performance.
    constexpr int kNumShmemElements = 256 / 64;

    __shared__ uint64_t shMasks[kNumShmemElements];
    __shared__ float shVals[kNumShmemElements];

    int index = threadIdx.x + blockDim.x * blockIdx.x;

    // Using bitwise shift for better performance in calculating planeIndex.
    int planeIndex = index >> 6;

    if (planeIndex >= n) return;

    // Use static_cast to ensure type correctness in accessing shared memory
    if (threadIdx.x < kNumShmemElements) {
        shMasks[threadIdx.x] = masks[planeIndex + threadIdx.x];
        shVals[threadIdx.x] = values[planeIndex + threadIdx.x];
    }
    __syncthreads();

    // Efficient mask and value retrieval from shared memory.
    uint64_t mask = shMasks[planeIndex];
    float val = shVals[planeIndex];

    // Use bitwise operation to determine if bit is set
    if ((mask & (1ull << (index & 0x3F))) != 0) {
        output[index] = val;
    } else {
        output[index] = 0;
    }
}