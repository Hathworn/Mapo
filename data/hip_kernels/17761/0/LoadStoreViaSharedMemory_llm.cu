#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_SHAREDSIZE 2048

__global__ void LoadStoreViaSharedMemory(int *In, int *Out)
{
#if 1
    int LoadStoreSize = MAX_SHAREDSIZE / blockDim.x;
    int beginIndex = threadIdx.x * LoadStoreSize;
    int endIndex = beginIndex + LoadStoreSize;

    // Optimize memory access by storing into registers first
    __shared__ int SharedMemory[MAX_SHAREDSIZE];
    int temp[MAX_SHAREDSIZE / blockDim.x];  // Use local memory for faster access
    int i;

    for (i = 0; i < LoadStoreSize; i++) {
        temp[i] = In[beginIndex + i];
    }

    __syncthreads();

    for (i = 0; i < LoadStoreSize; i++) {
        SharedMemory[beginIndex + i] = temp[i];  // Use shared memory for efficient access
    }

    __syncthreads();

    for (i = 0; i < LoadStoreSize; i++) {
        Out[beginIndex + i] = SharedMemory[beginIndex + i];
    }

    __syncthreads();
#else
    __shared__ int SharedMemory[MAX_SHAREDSIZE];

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    SharedMemory[idx] = In[idx];
    Out[idx] = SharedMemory[idx];
#endif
}