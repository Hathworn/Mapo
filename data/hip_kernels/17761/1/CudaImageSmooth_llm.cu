#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CudaImageSmooth(unsigned char *In, unsigned char *Out, int width, int height, int fsize)
{
    // Calculate column and row indices with block and thread info
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure col and row are within valid range
    if (col >= width || row >= height) return;
    
    // Calculate destination index
    int destIndex = row * width + col;

    // Initialize accumulator for smoothing
    float tmp = 0.0f;
    
    // Check boundary conditions
    if (col < fsize / 2 || col > width - fsize / 2 || row < fsize / 2 || row > height - fsize / 2) {
        Out[destIndex] = 0;
    } else {
        // Compute average pixel value within filter window
        for (int frow = -fsize / 2; frow <= fsize / 2; frow++) {
            for (int fcol = -fsize / 2; fcol <= fsize / 2; fcol++) {
                // Accumulate pixel values
                tmp += static_cast<float>(In[(row + frow) * width + (col + fcol)]);
            }
        }
        // Normalize by filter size to get average
        tmp /= (fsize * fsize);
        Out[destIndex] = static_cast<unsigned char>(tmp);
    }
}