#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16  // Define tile width for shared memory

__global__ void MatrixMult(int *M, int *N, int *P, int width)
{
    __shared__ int M_shared[TILE_WIDTH][TILE_WIDTH];  // Shared memory for sub-matrix of M
    __shared__ int N_shared[TILE_WIDTH][TILE_WIDTH];  // Shared memory for sub-matrix of N

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + tx;
    int row = blockIdx.y * TILE_WIDTH + ty;
    int Pv = 0;

    // Loop over tiles
    for (int m = 0; m < (width + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
        // Load M and N tiles into shared memory
        if (row < width && m * TILE_WIDTH + tx < width)
            M_shared[ty][tx] = M[row * width + m * TILE_WIDTH + tx];
        else
            M_shared[ty][tx] = 0;
        
        if (col < width && m * TILE_WIDTH + ty < width)
            N_shared[ty][tx] = N[(m * TILE_WIDTH + ty) * width + col];
        else
            N_shared[ty][tx] = 0;

        __syncthreads();  // Synchronize to ensure tiles are loaded
        
        // Accumulate results for this tile
        for (int k = 0; k < TILE_WIDTH; ++k) {
            Pv += M_shared[ty][k] * N_shared[k][tx];
        }

        __syncthreads();  // Synchronize before loading new tiles
    }

    // Write result to global memory
    if (row < width && col < width)
        P[row * width + col] = Pv;
}