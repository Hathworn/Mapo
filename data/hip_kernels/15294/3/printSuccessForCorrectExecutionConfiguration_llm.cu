#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void printSuccessForCorrectExecutionConfiguration()
{
    // Use more dynamic and less conditional print for efficiency
    printf(threadIdx.x == blockDim.x - 1 && blockIdx.x == gridDim.x - 1 ? "Success!\n" : "Failure. Update the execution configuration as necessary.\n");
}