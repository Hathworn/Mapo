#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initializeElementsTo(int initialValue, int *a, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // Ensure the index is within the bounds of the array
    if (i < N) {
        a[i] = initialValue;
    }
}