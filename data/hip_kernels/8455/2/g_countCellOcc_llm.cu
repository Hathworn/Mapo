#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void g_countCellOcc(uint *_hash, uint *_cellOcc, uint _pixCount, uint _hashCellCount)
{
    // Calculate the global thread index
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Avoid double calculation and reduce memory access
    if (idx < _pixCount) {
        uint hash_value = _hash[idx];
        if (hash_value < _hashCellCount) {
            atomicAdd(&(_cellOcc[hash_value]), 1);
        }
    }
}