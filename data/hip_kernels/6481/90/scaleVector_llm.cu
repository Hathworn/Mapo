#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize by using shared memory to reduce global memory access
__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    extern __shared__ float sharedData[];
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int localIdx = threadIdx.x;

    if (pos < len) {
        // Load data into shared memory
        sharedData[localIdx] = d_src[pos];
        __syncthreads();

        // Perform scaling operation
        d_res[pos] = sharedData[localIdx] * scale;
    }
}