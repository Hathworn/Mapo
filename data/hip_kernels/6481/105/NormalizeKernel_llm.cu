#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate the global thread index
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check to ensure threads are within matrix dimensions
    if (i < h && j < w) {
        int pos = i * s + j;
        float scale = normalization_factor[pos];
        
        // Use ternary operator to handle division by zero
        float invScale = (scale != 0.0f) ? 1.0f / scale : 1.0f; 

        image[pos] *= invScale; // Apply normalization
    }
}