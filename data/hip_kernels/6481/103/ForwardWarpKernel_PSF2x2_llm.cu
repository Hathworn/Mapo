#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Pre-calculate common expressions
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);

    float value = src[image_row_offset + j];
    float weight;

    // Use inline functions for atomicAdd operations
    auto safe_atomic_add = [&](int tx, int ty, float weight) {
        if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
            _atomicAdd(dst + ty * image_stride + tx, value * weight);
            _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
        }
    };

    // Fill four corners with appropriate weights
    safe_atomic_add(tx, ty, dx * dy);                       // Bottom right corner
    safe_atomic_add(tx - 1, ty, (1.0f - dx) * dy);          // Bottom left corner
    safe_atomic_add(tx - 1, ty - 1, (1.0f - dx) * (1.0f - dy)); // Upper left corner
    safe_atomic_add(tx, ty - 1, dx * (1.0f - dy));          // Upper right corner
}