#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void squareFunc(unsigned int *d_in, unsigned int *d_out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Ensure unique thread index across grid
    if (idx < blockDim.x) { // Boundary check to avoid out-of-bounds access
        unsigned int val = d_in[idx];
        d_out[idx] = val * val;
    }
    //printf("%d square value %d \n  ", idx, d_out[idx]);
}