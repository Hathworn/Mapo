#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_hello() {
    // Use thread 0 of block 0 to avoid excessive printing
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Hello World from GPU!\n");
    }
}