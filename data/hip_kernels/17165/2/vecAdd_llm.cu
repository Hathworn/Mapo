#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(double *a, double *b, double *c, int n) {
    // Calculate contiguous global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Improved bounds check with efficient memory access
    if (id < n) {
        double temp_a = a[id];  // Cache data from global memory
        double temp_b = b[id];  // Cache data from global memory
        c[id] = temp_a + temp_b; // Perform addition and write to global memory
    }
}