#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(double *a, double *b, double *c, int n) {
    // Use grid-stride loops for scalability
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = id; i < n; i += blockDim.x * gridDim.x) {
        c[i] = a[i] + b[i];
    }
}