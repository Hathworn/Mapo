#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(double *a, double *b, double *c, int n) {
    // Use shared memory to reduce global memory access latency
    extern __shared__ double sharedData[];

    int tid = threadIdx.x;
    int id = blockIdx.x * blockDim.x + tid;

    // Load elements into shared memory
    if (id < n) {
        sharedData[tid] = a[id] + b[id];
        __syncthreads();
        
        // Write results back to global memory
        c[id] = sharedData[tid];
    }
}