#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_renorm(float *data, const float value, const ptrdiff_t size, const float maxnorm)
{
    __shared__ float buffer[32];
    long tx = threadIdx.x;
    long bx = blockIdx.x;
    long step = blockDim.x;
    float *row = data + size * bx;

    buffer[tx] = 0;

    // Calculate norm of axis using stride loop
    for (ptrdiff_t i = tx; i < size; i += step)
    {
        buffer[tx] += powf(fabsf(row[i]), value); // Use powf and fabsf for better performance
    }

    // Sum reduction using shared memory and unrolling
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        __syncthreads();
        if (tx < stride)
        {
            buffer[tx] += buffer[tx + stride];
        }
    }

    // Clip norms
    __syncthreads();
    float norm = powf(buffer[0], 1 / value); // Use powf for better performance
    if (norm > maxnorm)
    {
        norm = maxnorm / (norm + 1e-7);
        // Renormalize
        for (ptrdiff_t i = tx; i < size; i += step)
        {
            row[i] *= norm;
        }
    }
}