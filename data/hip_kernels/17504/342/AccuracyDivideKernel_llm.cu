#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Optimize the division operation by using atomic operations
    // This ensures that the accuracy is only updated once accurately by threads
    if (threadIdx.x == 0) {  // Ensure only one thread modifies the value
        atomicExch(accuracy, *accuracy / N);
    }
}