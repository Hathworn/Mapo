#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void read_G_matrix_kernel(int S, int vertex_index, int* i_index, int* j_index, bool* is_Bennett, double* exp_Vj, double* N_ptr, int LD_N, double* G_ptr, int LD_G, double* result_ptr, int incr) {
    int l = threadIdx.x;
    
    // Pre-fetch reused values and pre-compute conditions
    int j_idx = j_index[l];
    int i_idx = i_index[l];
    double exp_val = exp_Vj[l];
    bool bennett_cond = is_Bennett[l];
    
    double result, delta;
    
    if (j_idx < vertex_index) {
        delta = (i_idx == j_idx) ? 1.0 : 0.0;
        // Simplified calculation
        result = (N_ptr[i_idx + LD_N * j_idx] * exp_val - delta) / (exp_val - 1.0);
    } else {
        result = G_ptr[i_idx + LD_G * (j_idx - vertex_index)];
    }
    
    // Simplified conditional assignment
    result_ptr[l * incr] = bennett_cond ? 0.0 : result;
}