#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_col_on_Gamma_matrix_kernel(int col_index, int vertex_index, int* indices, double* exp_V, double* N_ptr, int LD_N, double* G_ptr, int LD_G, double* col_ptr, int incr) {
    // Use multiple threads in a block to process multiple elements concurrently
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if the thread index is within the problem size
    if(tid < vertex_index) {
        int i_index = indices[tid];
        int j_index = indices[col_index];
        double exp_Vj = exp_V[col_index];
        
        if (j_index < vertex_index) {
            double delta = (i_index == j_index) ? 1.0 : 0.0;
            col_ptr[tid * incr] = (N_ptr[i_index + LD_N * j_index] * exp_Vj - delta) / (exp_Vj - 1.0);
        } else {
            col_ptr[tid * incr] = G_ptr[i_index + LD_G * (j_index - vertex_index)];
        }
    }
}