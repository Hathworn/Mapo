#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _segmentedScanBackKer(float *maxdist, int *maxdistidx, int *label, float *blockmaxdist, int *blocklabel, int *blockmaxdistidx, int numelements) {
    // Declare shared memory for storing the last element of the previous block
    __shared__ float shdcurmaxdist;
    __shared__ int shdcurlabel;
    __shared__ int shdcurmaxdistindex;
    __shared__ int state;

    // Calculate the index for block-level accumulation
    int idx = (blockIdx.x + 1) * blockDim.x + threadIdx.x;

    // First thread of each block reads the previous block's last element
    if (threadIdx.x == 0) {
        shdcurmaxdist = blockmaxdist[blockIdx.x];
        shdcurlabel = blocklabel[blockIdx.x];
        shdcurmaxdistindex = blockmaxdistidx[blockIdx.x];
        state = (label[idx] == shdcurlabel);
    }

    // Synchronize within the block
    __syncthreads();

    // Exit if the previous block is unrelated, or index is beyond array length
    if (state == 0 || idx >= numelements)
        return;

    // Update max distance and index if the segment label matches
    if (label[idx] == shdcurlabel && maxdist[idx] < shdcurmaxdist) {
        maxdist[idx] = shdcurmaxdist;
        maxdistidx[idx] = shdcurmaxdistindex;
    }
}