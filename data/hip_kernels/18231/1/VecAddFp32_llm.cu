#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAddFp32(float* in0, float* in1, float* out, int cnt)
{
    // Calculate the index for the current thread
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure out-of-bounds memory access is prevented
    if (tid < cnt) {
        // Utilize shared memory for coalesced access and minimizing latency
        __shared__ float temp0, temp1;
        temp0 = in0[tid];
        temp1 = in1[tid];
        
        // Perform the addition and store in output
        out[tid] = temp0 + temp1;
    }
}