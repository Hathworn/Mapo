#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAddInt32(int32_t* __restrict__ in0, int32_t* __restrict__ in1, int32_t* __restrict__ out, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Loop over all elements in the stride using the grid-stride loop
    for (int i = tid; i < cnt; i += stride) {
        out[i] = in0[i] + in1[i];
    }
}