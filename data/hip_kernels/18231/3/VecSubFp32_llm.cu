#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecSubFp32(float* in0, float* in1, float* out, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to improve data access speed
    __shared__ float s_in0[256];
    __shared__ float s_in1[256];
    
    if (tid < cnt) {
        s_in0[threadIdx.x] = in0[tid];
        s_in1[threadIdx.x] = in1[tid];
        __syncthreads(); // Ensure all threads have loaded data into shared memory

        out[tid] = s_in0[threadIdx.x] - s_in1[threadIdx.x];
    }
}