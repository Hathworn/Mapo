#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_rng (hiprandState * states, int seed)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;
hiprand_init (seed, tid, 0, &states[tid]);
}