#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_rng(hiprandState *states, int seed) {
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize RNG for each thread
    hiprandState localState;
    hiprand_init(seed, tid, 0, &localState);
    states[tid] = localState;
}