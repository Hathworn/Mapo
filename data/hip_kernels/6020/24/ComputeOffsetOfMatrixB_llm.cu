#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Calculate global thread index
    int32_t globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    // Stride to handle larger arrays
    int32_t stride = blockDim.x * gridDim.x;
    for (int32_t i = globalIdx; i < N; i += stride) {
        output[blockIdx.x * N + i] = -row_sum[blockIdx.x];
    }
}