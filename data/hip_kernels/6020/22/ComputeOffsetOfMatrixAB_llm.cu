#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixAB(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, int32_t K_A_B, int32_t N) {
    // Calculate global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (idx < N) {
        // Optimized computation with pre-computed indexes
        int32_t value = K_A_B - row_sum[blockIdx.x] - col_sum[idx];
        output[blockIdx.x * N + idx] = value;
    }
}