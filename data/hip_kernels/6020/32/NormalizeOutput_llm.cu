#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a while loop for improved performance by reducing control overhead
    while (idx < num_elements) {
        int64_t* output_ptr = to_normalize + idx * 3; // Pre-calculate the output address
        output_ptr[0] = batch_index;
        output_ptr[1] = class_index;
        output_ptr[2] = static_cast<int64_t>(original[idx]);
        idx += blockDim.x * gridDim.x;
    }
}