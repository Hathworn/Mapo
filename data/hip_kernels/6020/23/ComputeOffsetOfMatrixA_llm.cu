#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    int32_t globalIdx = blockDim.x * blockIdx.x + threadIdx.x; // Calculate global index
    if (globalIdx < N) { // Check index range
        output[blockIdx.x * N + globalIdx] = -col_sum[globalIdx]; // Direct assignment
    }
}