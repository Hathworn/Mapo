#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int addem(int a, int b) {
    return a + b;
}

__global__ void add(int a, int b, int *c) {
    // Use shared memory for intermediate storage to optimize memory access
    extern __shared__ int shared_mem[];
    int index = threadIdx.x;
    shared_mem[index] = addem(a, b);

    // Use a single thread to store the result back to global memory
    if (index == 0) {
        *c = shared_mem[0];
    }
}