#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(unsigned char *ptr, int ticks) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    
    // Pre-compute constants outside the loop
    const float dim_half = DIM / 2.0f;
    const float ticks_factor = ticks / 7.0f;

    // now calculate the value at that position
    float fx = x - dim_half;
    float fy = y - dim_half;
    float d = sqrtf(fx * fx + fy * fy);
    float d_div_10 = d / 10.0f;
    
    // Optimize the cosine and division computation
    float grey_f = 128.0f + 127.0f * cosf(d_div_10 - ticks_factor) / (d_div_10 + 1.0f);
    unsigned char grey = static_cast<unsigned char>(grey_f);
    
    ptr[offset * 4 + 0] = grey;
    ptr[offset * 4 + 1] = grey;
    ptr[offset * 4 + 2] = grey;
    ptr[offset * 4 + 3] = 255;
}