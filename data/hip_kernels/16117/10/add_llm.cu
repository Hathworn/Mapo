#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel for better performance
__global__ void add(int a, int b, int *c) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global thread ID
    if (tid == 0) { // Ensure only one thread performs the operation
        *c = a + b;
    }
}