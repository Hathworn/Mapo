#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel( uchar4 *ptr, int ticks ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // calculate fx and fy only once
    float fx = x - DIM / 2;
    float fy = y - DIM / 2;
    
    // optimize distance calculation by reducing expensive sqrt operation
    float d_sq = fx * fx + fy * fy;
    float d = sqrtf(d_sq);

    // use a register to store intermediate calculation for grey
    float cos_val = cosf(d / 10.0f - ticks / 7.0f);
    unsigned char grey = (unsigned char)(128.0f + 127.0f * cos_val / (d / 10.0f + 1.0f));

    // use vectorized assignment
    ptr[offset] = make_uchar4(grey, grey, grey, 255);
}