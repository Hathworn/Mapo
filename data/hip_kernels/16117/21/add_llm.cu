#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < N; i += stride) { // Use a for loop for better readability
        c[i] = a[i] + b[i]; // Perform add operation
    }
}