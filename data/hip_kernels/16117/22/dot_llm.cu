#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(float *a, float *b, float *c) {
    // Use dynamic shared memory allocation
    extern __shared__ float cache[];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0f;
    // Unroll the loop for better performance
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        temp += a[i] * b[i];
    }

    // Cache the partial sum
    cache[cacheIndex] = temp;

    // Synchronize threads in this block
    __syncthreads();

    // Reduce within shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (cacheIndex < stride) {
            // Use atomic addition for better performance
            cache[cacheIndex] += cache[cacheIndex + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}