#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int* c)
{
    // Using shared memory to cache input values
    __shared__ int a_shared, b_shared;
    int tid = threadIdx.x;

    // Load data into shared memory
    if (tid == 0) {
        a_shared = a;
        b_shared = b;
    }
    __syncthreads();

    // Perform addition using shared memory values
    if (tid == 0) {
        *c = a_shared + b_shared;
    }
}