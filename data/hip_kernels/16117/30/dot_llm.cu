#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(int size, float *a, float *b, float *c) {
    extern __shared__ float cache[];  // Use dynamic shared memory
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0f;
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        temp += a[i] * b[i];  // Loop unrolling
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    // Reduction using loop unrolling
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}