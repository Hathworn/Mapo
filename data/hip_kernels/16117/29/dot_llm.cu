#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(int size, float *a, float *b, float *c) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;

    // Unroll the loop to reduce overhead and increase performance
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        temp += a[i] * b[i];
    }

    // Assign the computed value to shared memory
    cache[cacheIndex] = temp;

    // Synchronize threads in this block
    __syncthreads();

    // Reduce using binary tree reduction
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    // Write the computed block result to global memory
    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}