#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histo_kernel( unsigned char *buffer, long size, unsigned int *histo ) {
    // Use a register to store the temporary count
    __shared__ unsigned int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Unrolling the loop for better performance
    while (i < size) {
        atomicAdd(&temp[buffer[i]], 1);
        if (i + stride < size) {
            atomicAdd(&temp[buffer[i + stride]], 1);
        }
        i += 2 * stride; // Unroll loop by factor of 2
    }

    __syncthreads();

    // Use warp reduction for faster accumulation
    unsigned int sum = temp[threadIdx.x];
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xffffffff, sum, offset);
    }

    if (threadIdx.x % warpSize == 0) {
        temp[threadIdx.x / warpSize] = sum;
    }

    __syncthreads();

    if (threadIdx.x < blockDim.x / warpSize) {
        sum = temp[threadIdx.x];
        for (int offset = blockDim.x / (2 * warpSize); offset > 0; offset /= 2) {
            sum += __shfl_down_sync(0xffffffff, sum, offset);
        }
        if (threadIdx.x == 0) {
            atomicAdd(&histo[blockIdx.x], sum);
        }
    }
}