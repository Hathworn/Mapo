#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo) {
    // Use shared memory for histogram bins
    __shared__ unsigned int localHisto[256];
    if (threadIdx.x < 256) {
        localHisto[threadIdx.x] = 0; // Initialize shared memory bins
    }
    __syncthreads();

    // Calculate the starting index and the offset to the next block each thread will process
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while (i < size) {
        atomicAdd(&localHisto[buffer[i]], 1); // Update shared histogram
        i += stride;
    }
    __syncthreads();
    
    // Write back to global memory
    if (threadIdx.x < 256) {
        atomicAdd(&histo[threadIdx.x], localHisto[threadIdx.x]);
    }
}