#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c) {
    // Calculate unique thread index across the grid
    int tid = blockIdx.x * blockDim.x + threadIdx.x; 
    // Use the unique thread index for computation
    if (tid < N) { 
        c[tid] = a[tid] + b[tid]; // Perform addition
    }
}