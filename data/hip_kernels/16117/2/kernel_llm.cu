#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel( int *a, int *b, int *c, int N ) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        // Use local variables to reduce global memory access
        int local_idx1 = (idx + 1) & 255; 
        int local_idx2 = (idx + 2) & 255;
        float a_sum = a[idx] + a[local_idx1] + a[local_idx2];
        float b_sum = b[idx] + b[local_idx1] + b[local_idx2];
        c[idx] = (a_sum + b_sum) / 6.0f; // Direct calculation to avoid redundant division
    }
}