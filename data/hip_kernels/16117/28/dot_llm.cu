#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define threadsPerBlock 256

__global__ void dot(int size, float *a, float *b, float *c) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0f;
    
    // Efficient loop iterating to cover the input size
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        temp += a[i] * b[i];
    }

    cache[cacheIndex] = temp;

    __syncthreads();

    // Use a more efficient reduction with unrolled loop
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}