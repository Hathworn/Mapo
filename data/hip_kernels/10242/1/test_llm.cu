#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void test(float *a, float *b, float *c, int N)
{
    // Use threadIdx and blockDim for parallel computation
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Only compute within the bounds of N
    if (idx < N)
    {
        c[idx] = a[idx] * b[idx];
    }
}