#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Use faster approximation for pow function
    float base = 3.14159;

    for (int i = tid; i < n; i += stride) {
        x[i] = sqrt(__powf(base, i)); // Use __powf for higher performance
    }
}