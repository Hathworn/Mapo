#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void test(float *a, float *b, float *c, int N)
{
    // Calculate unique global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure the thread operates within array bounds
    if(idx < N)
    {
        c[idx] = a[idx] * b[idx];
    }
    return;
}