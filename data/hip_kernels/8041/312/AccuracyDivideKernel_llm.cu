#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory for local storage to reduce global memory access
    __shared__ float local_accuracy;

    // Thread 0 initializes the shared memory variable
    if (threadIdx.x == 0) {
        local_accuracy = *accuracy;
    }

    // Sync threads to ensure shared memory is initialized
    __syncthreads();

    // Thread 0 performs the division and updates the global variable
    if (threadIdx.x == 0) {
        local_accuracy /= N;
        *accuracy = local_accuracy;
    }
}