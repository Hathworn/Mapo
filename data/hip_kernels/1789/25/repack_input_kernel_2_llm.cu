#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void repack_input_kernel_2(float *input, float *re_packed_input, int w, int h, int c)
{
    // Calculate index for parallel work across threads
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    const int items_per_channel = w * h;

    // Precompute values for efficient access
    int c_pack = index % 32;
    int chan_index = index / 32;
    int chan = (chan_index * 32) % c;
    int i = (chan_index * 32) / c;

    // Check bounds to avoid out of range access
    if (i < items_per_channel && chan + c_pack < c) 
    {
        // Accessing input and repacking without looping
        float src = input[(chan + c_pack) * items_per_channel + i];
        re_packed_input[chan * items_per_channel + i * 32 + c_pack] = src;
    }
}