#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void repack_input_kernel(float *input, float *re_packed_input, int w, int h, int c)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    const int items_per_channel = w * h;
    
    // Derive channel and item indices
    int c_pack = index % 32;
    int chan_index = index / 32;
    int chan = chan_index % (c / 32) * 32;
    int i = chan_index / (c / 32);
    
    if (chan < c && i < items_per_channel)
    {
        // Optimize memory access pattern
        float src = input[(chan + c_pack) * items_per_channel + i];
        re_packed_input[chan * items_per_channel + i * 32 + c_pack] = src;
    }
}