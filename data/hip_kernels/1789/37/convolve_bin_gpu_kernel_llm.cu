#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __host__ static inline uint8_t xnor_bit1(uint8_t a, uint8_t b) {
    return ~(a^b) & 0b1;
}

__device__ __host__ static inline unsigned char get_bit(unsigned char const* const src, size_t index) {
    size_t src_i = index / 8;
    int src_shift = index % 8;
    unsigned char val = (src[src_i] & (1 << src_shift)) > 0;
    return val;
}

__global__ void convolve_bin_gpu_kernel(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n, int size, int pad, int new_lda, float *mean_arr_gpu) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    int x = index % in_w;
    int index2 = index / in_w;
    int y = index2 % in_h;
    int fil = index2 / in_h;

    if (fil < n) {
        int const output_index = fil * in_w * in_h + y * in_w + x;
        int sum = 0;
        int good_val = 0;
        
        int min_index = blockIdx.x * blockDim.x;
        int min_fil = (min_index / in_w) / in_h;
        int max_index = (blockIdx.x + 1) * blockDim.x - 1;
        int max_fil = (max_index / in_w) / in_h;

        __shared__ uint32_t weights_shared[3 * 3 * 1024 * 6 / 32 + 1];
        const int weights_size = size * size * in_c / 32 + 1;

        for (int tmp_fil = min_fil; tmp_fil <= max_fil; tmp_fil++) {
            for (int s = threadIdx.x; s < weights_size; s += blockDim.x) {
                weights_shared[s + (tmp_fil - min_fil) * new_lda / 32] = ((uint32_t *)weights)[tmp_fil * new_lda / 32 + s];
            }
        }
        __syncthreads();

        __shared__ uint32_t input_shared[416 * 416 / 32 + 1];
        const int input_shared_size = in_w * in_h / 32 + 1;
        for (int s = threadIdx.x; s < input_shared_size; s += blockDim.x) {
            input_shared[s] = ((uint32_t *)input)[chan * in_w * in_h / 32 + s];
        }
        __syncthreads();

        for (int chan = 0; chan < in_c; ++chan) {
            int const input_pre_index = chan * in_w * in_h;
            int input_y, input_x;
            for (int f_y = 0; f_y < size; ++f_y) {
                input_y = y + f_y - pad;
                for (int f_x = 0; f_x < size; ++f_x) {
                    input_x = x + f_x - pad;
                    if (input_y < 0 || input_x < 0 || input_y >= in_h || input_x >= in_w) continue;

                    int weights_shared_index = (fil - min_fil) * new_lda + chan * size * size + f_y * size + f_x;
                    uint8_t w_bit = get_bit((uint8_t *)weights_shared, weights_shared_index);

                    int input_shared_index = input_y * in_w + input_x;
                    uint8_t in_bit = get_bit((uint8_t *)input_shared, input_shared_index);

                    int res = xnor_bit1(in_bit, w_bit);
                    sum += res;
                    good_val++;
                }
            }
        }
        sum = sum - (good_val - sum);
        atomicAdd(&output[output_index], sum * mean_arr_gpu[fil]);
    }
}