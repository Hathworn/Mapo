#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_weights_mean_kernel(float *weights, int n, int size, float *binary, float *mean_arr_gpu)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int f = i / size;

    // Check if index is within bounds
    if (i < n * size) {
        float mean = mean_arr_gpu[f];
        binary[i] = (weights[i] > 0) ? mean : -mean;
    }
}