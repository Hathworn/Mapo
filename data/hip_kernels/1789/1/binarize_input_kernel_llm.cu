#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = blockIdx.x * blockDim.x + threadIdx.x; // Use only blockIdx.x for 1D grid
    if (s >= size) return;
    float mean = 0.0f;

    // Replace division within the loop 
    for(int i = 0; i < n; ++i){
        mean += fabsf(input[i * size + s]); // Use fabsf for float precision
    }
    
    mean /= n; // Perform division once after loop

    // Use same loop condition to improve data locality
    for(int i = 0; i < n; ++i){
        binary[i * size + s] = copysignf(mean, input[i * size + s]); // Use copysignf for clarity
    }
}