#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Calculate index using grid-stride loop for better workload distribution
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        binary[i] = (x[i] >= 0) ? 1 : -1;
    }
}