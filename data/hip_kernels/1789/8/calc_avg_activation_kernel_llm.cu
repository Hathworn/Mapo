#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calc_avg_activation_kernel(float *src, float *dst, int size, int channels, int batches)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size * batches) return; // Early exit if index is out of range

    int xy = i % size;
    int b = i / size;

    float sum = 0;
    // Use a temporary variable to accumulate results
    for (int c = 0; c < channels; ++c) {
        sum += src[xy + size * (c + channels * b)];
    }
    // Assign the average to the destination array
    dst[i] = sum / channels;
}