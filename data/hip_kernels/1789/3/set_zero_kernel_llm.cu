#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_zero_kernel(float *src, int size)
{
    // Use a stride loop to allow for fewer blocks
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        src[i] = 0;
    }
}