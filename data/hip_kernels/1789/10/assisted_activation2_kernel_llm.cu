#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void assisted_activation2_kernel(float alpha, float *output, float *gt_gpu, float *a_avg_gpu, int size, int channels, int batches)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int b = i / size;
    float beta = 1 - alpha;

    // Optimize: Check bounds before calculations inside loop
    if (b < batches) {
        int xy = i % size;
        for (int c = 0; c < channels; ++c) {
            // Optimize: Pre-calculate index for reuse
            int index = xy + size * (c + channels * b);
            if (gt_gpu[index] == 0) {
                output[index] *= beta;
            }
        }
    }
}