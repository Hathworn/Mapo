#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    int b = id / c;

    int out_index = k + c * b;

    // Initialize sum accumulator
    float sum = 0.0f;
    
    // Compute input base index for the current output element
    int input_base_index = h * w * (k + b * c);

    // Accumulate sum for the [w*h] elements related to the current output
    for(int i = 0; i < w * h; ++i) {
        sum += input[input_base_index + i];
    }

    // Compute the average and store it in the output
    output[out_index] = sum / (w * h);
}