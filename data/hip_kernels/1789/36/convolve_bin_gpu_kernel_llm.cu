#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __host__ static inline uint8_t xnor_bit1(uint8_t a, uint8_t b) {
    return ~(a^b) & 0b1;
}

__device__ __host__ static inline unsigned char get_bit(unsigned char const* const src, size_t index) {
    size_t src_i = index / 8;
    int src_shift = index % 8;
    unsigned char val = (src[src_i] & (1 << src_shift)) > 0;
    return val;
}

__global__ void convolve_bin_gpu_kernel(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n, int size, int pad, int new_lda, float *mean_arr_gpu) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    int x = index % in_w;
    int index2 = index / in_w;
    int y = index2 % in_h;
    int fil = index2 / in_h;

    if (fil < n) {
        int const output_index = fil * in_w * in_h + y * in_w + x;
        int sum = 0;
        int good_val = 0;
        
        // Iterate over channels
        for (int chan = 0; chan < in_c; ++chan) {
            int const weights_pre_index = fil * new_lda + chan * size * size;
            int const input_pre_index = chan * in_w * in_h;

            // Iterate over filter dimensions
            for (int f_y = 0; f_y < size; ++f_y) {
                int input_y = y + f_y - pad;
                if (input_y < 0 || input_y >= in_h) continue;  // Boundary check for y

                for (int f_x = 0; f_x < size; ++f_x) {
                    int input_x = x + f_x - pad;
                    if (input_x < 0 || input_x >= in_w) continue;  // Boundary check for x

                    int input_index = input_pre_index + input_y * in_w + input_x;
                    int weights_index = weights_pre_index + f_y * size + f_x;

                    uint8_t in_bit = get_bit((uint8_t *)input, input_index);
                    uint8_t w_bit = get_bit((uint8_t *)weights, weights_index);
                    int res = xnor_bit1(in_bit, w_bit);
                    sum += res;
                    good_val++;
                }
            }
        }
        
        // Adjust sum and store in output
        sum = 2 * sum - good_val;
        output[output_index] = sum * mean_arr_gpu[fil];  // Potential atomicAdd for inter-BLOCK sum
    }
}