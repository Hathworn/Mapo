#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int total_size = w * h;
    int k = id % c;
    int b = id / c; // Use integer division for batch index
    int out_index = k + c * b;
    float scale = out_delta[out_index] / total_size;

    int in_base_index = k + b * c;
    for(int i = threadIdx.y; i < total_size; i += blockDim.y) {
        int in_index = i + h * w * in_base_index; // Calculate index once per loop iteration
        in_delta[in_index] += scale;
    }
}