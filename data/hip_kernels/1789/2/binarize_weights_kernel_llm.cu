#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
    
    // Use shared memory to reduce global memory access
    extern __shared__ float shared_weights[];
    
    int idx = threadIdx.x;
    float mean = 0;
    
    // Load data into shared memory
    for (int i = idx; i < size; i += blockDim.x) {
        shared_weights[i] = fabs(weights[f * size + i]);
    }
    __syncthreads();
    
    // Compute the mean using shared memory
    for (int i = idx; i < size; i += blockDim.x) {
        atomicAdd(&mean, shared_weights[i]);
    }
    
    // Calculate mean
    if (idx == 0) {
        mean /= size;
    }
    __syncthreads();
    
    // Binarize weights
    for (int i = idx; i < size; i += blockDim.x) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    }
}