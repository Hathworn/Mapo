#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_uint32_kernel_2(uint32_t *src, uint32_t *dst, int src_h, int src_w, int src_align, int dst_align)
{
    __shared__ uint32_t tmp[33 * 32];   // Shared memory with padding

    const int w_align = 33;
    const int src_w_align = src_w + (32 - src_w % 32);

    const int warps_in_width = src_w_align / 32;

    const int local_x = threadIdx.x % 32;   // Local x in warp
    const int local_y = threadIdx.x / 32;   // Local y in warp

    const int global_index = blockIdx.x;
    const int global_x_index = global_index % warps_in_width;
    const int global_y_index = global_index / warps_in_width;

    const int global_x = global_x_index * 32 + local_x;
    const int global_y = global_y_index * 32 + local_y;

    uint32_t val = 0;
    if (global_x < src_w && global_y < src_h) {
        val = src[global_y * src_align + global_x];
    }

    // Coalesce global reads by transposing in shared memory
    tmp[local_y * w_align + local_x] = val;
    __syncthreads();

    // Read transposed data from shared memory
    val = tmp[local_x * w_align + local_y];

    const int new_global_x = global_y_index * 32 + local_x;
    const int new_global_y = global_x_index * 32 + local_y;

    if (new_global_x < src_h && new_global_y < src_w) {
        dst[new_global_y * (dst_align / 32) + new_global_x] = val;
    }
}