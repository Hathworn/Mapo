#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void assisted_activation_kernel(float alpha, float *output, float *gt_gpu, float *a_avg_gpu, int size, int channels, int batches)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size * batches) return; // Early exit if index exceeds limit

    int xy = i % size;
    int b = i / size;

    for (int c = 0; c < channels; ++c) {
        int idx = xy + size * (c + channels * b); // Compute index outside loop
        float gt_val = gt_gpu[i]; // Load reused values outside the inner loop
        float a_avg_val = a_avg_gpu[i];
        output[idx] += alpha * gt_val * a_avg_val;
    }
}