#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_uint32_kernel(uint32_t *src, uint32_t *dst, int src_h, int src_w, int src_align, int dst_align)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Determine row and column from index
    int i = index % src_h;
    int j = index / src_h;

    // Perform write only if within bounds
    if (j < src_w)
    {
        // Improved memory access pattern by reusing calculated index
        dst[j * (dst_align / 32) + i] = src[i * src_align + j];
    }
}