#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xcorr(float *d_i1, float *d_i2, float *d_icorr, int m1, int n1, int m2, int n2)
{
    // Thread and block indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row_out = blockIdx.y * blockDim.y + ty;
    int col_out = blockIdx.x * blockDim.x + tx;
    int row_st = row_out - (m2 - 1);
    int col_st = col_out - (n2 - 1);

    float corr = 0; // Initialize correlation variable

    // Only process valid output locations
    if (row_out >= 0 && row_out < m1 + m2 - 1 && col_out >= 0 && col_out < n1 + n2 - 1)
    {
        // Optimize memory access by reducing condition checks
        #pragma unroll
        for (int i = 0; i < m2; i++)
        {
            #pragma unroll
            for (int j = 0; j < n2; j++)
            {
                int d_i1_row = row_st + i;
                int d_i1_col = col_st + j;
                if (d_i1_row >= 0 && d_i1_row < m1 && d_i1_col >= 0 && d_i1_col < n1)
                {
                    // Perform convolution
                    corr += d_i1[d_i1_row + d_i1_col * m1] * d_i2[i + j * m2];
                }
            }
        }
        // Assign correlation variable to proper location in final image
        d_icorr[row_out + col_out * (m1 + m2 - 1)] = corr;
    }
}