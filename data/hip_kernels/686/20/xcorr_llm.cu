#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xcorr(float *d_i1, float *d_i2, float *d_icorr, int m1, int n1, int m2, int n2)
{
    //grab threadID location
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // get output values based on block and thread locations
    int row_out = blockIdx.y * blockDim.y + ty;
    int col_out = blockIdx.x * blockDim.x + tx;

    // Get starting value for the convolution as dictated by m2 and n2
    int row_st = row_out - (m2 - 1);
    int col_st = col_out - (n2 - 1);

    // correlation variable
    float corr = 0.0f;  // initialize correlation variable

    if (row_out >= 0 && row_out < m1 + m2 - 1 && col_out >= 0 && col_out < n1 + n2 - 1)  // ensure output is within bounds of correlation image
    {
        for (int i = 0; i < m2; i++) 
        {
            for (int j = 0; j < n2; j++)
            {
                int ii = row_st + i;
                int jj = col_st + j;
                // Check bounds and compute correlation
                if (ii >= 0 && ii < m1 && jj >= 0 && jj < n1) 
                {
                    corr += d_i1[ii + jj * m1] * d_i2[i + j * m2];
                }
            }
        }
        d_icorr[row_out + col_out * (m1 + m2 - 1)] = corr;  // assign correlation variable to proper location in final image
    }
}