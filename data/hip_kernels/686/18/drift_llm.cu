#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void drift(float *d_i1, float *d_icorr, int m1, int n1, int o1)
{
    //grab theadID location
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // get output values based on block and thread locations
    int row_out = blockIdx.y * blockDim.y + ty;
    int col_out = blockIdx.x * blockDim.x + tx;
    int im_out  = blockIdx.z;

    // Get starting value for the convolution as dictated by m2 and n2
    // we'll use i1 indicies as the coord syst.
    int row_st = row_out - (m1 - 1);
    int col_st = col_out - (n1 - 1);

    // correlation variable
    float corr = 0; // initialize correlation variable

    if (row_out >= 0 && row_out < 2 * m1 - 1 && col_out >= 0 && col_out < 2 * n1 - 1 && im_out < o1 - 1) // ensure output is within bounds of correlation image
    {
        // Unrolling the inner loop would help in parallel computation
        #pragma unroll
        for (int i = 0; i < m1; i++) {
            #pragma unroll
            for (int j = 0; j < n1; j++)
            {
                if (row_st + i >= 0 && row_st + i < m1 && col_st + j >= 0 && col_st + j < n1) { // check bounds
                    corr += d_i1[row_st + i + (col_st + j) * m1 + im_out * m1 * n1] *
                            d_i1[i + j * m1 + (im_out + 1) * m1 * n1]; // shift n+1 image over n image
                }
            }
        }
        d_icorr[row_out + col_out * (2 * m1 - 1) + im_out * (2 * m1 - 1) * (2 * n1 - 1)] = corr; // store result
    }
}