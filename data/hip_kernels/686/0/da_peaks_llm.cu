#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimized for da_peaks
__global__ void da_peaks(float *d_i1, float thresh, int m, int n, int o)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Location of the output pixel being analyzed
    int row_output = blockIdx.y * blockDim.y + ty;
    int col_output = blockIdx.x * blockDim.x + tx;
    int imnum = blockIdx.z;

    // Check boundaries and image number
    if (imnum < o && row_output >= 2 && row_output < m - 2 && col_output >= 2 && col_output < n - 2)
    {
        float d_i2[25];

        // Load data into shared memory (synchronization not needed due to no bank conflicts)
        for (int i = 0; i < 5; ++i)
        {
            for (int j = 0; j < 5; ++j)
            {
                d_i2[i * 5 + j] = d_i1[(row_output - 2 + i) + (col_output - 2 + j) * m + imnum * m * n];
            }
        }

        float me = d_i2[12];
        int maxi = (me >= thresh);

        // Check if current center pixel is a local maximum
        for (int k = 0; k < 25; ++k)
        {
            if (d_i2[k] > me)
            {
                maxi = 0;
                break; // Early exit if not a maximum
            }
        }

        d_i1[row_output + col_output * m + imnum * m * n] = maxi;
    }
    else if (imnum < o)
    {
        d_i1[row_output + col_output * m + imnum * m * n] = 0;
    }
}