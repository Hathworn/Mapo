#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void xcorr(float *d_i1, float *d_i2, float *d_icorr, int m1, int n1, int m2, int n2)
{
    // Thread indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Output location indices
    int row_out = blockIdx.y * blockDim.y + ty;
    int col_out = blockIdx.x * blockDim.x + tx;

    // Starting indices for convolution
    int row_st = row_out - (m2 - 1);
    int col_st = col_out - (n2 - 1);

    // Initialize correlation variable
    float corr = 0;

    // Check bounds for output
    if (row_out < m1 + m2 - 1 && col_out < n1 + n2 - 1 && row_out >= 0 && col_out >= 0) 
    {
        // Iterate over the kernel dimensions
        for (int i = 0; i < m2; i++) 
        {
            for (int j = 0; j < n2; j++)
            {
                // Ensure indices are within bounds of image
                int row_idx = row_st + i;
                int col_idx = col_st + j;
                if (row_idx >= 0 && row_idx < m1 && col_idx >= 0 && col_idx < n1) 
                {
                    corr += d_i1[row_idx + col_idx * m1] * d_i2[i + j * m2];
                }
            }
        }
        // Write the computed correlation to the output
        d_icorr[row_out + col_out * (m1 + m2 - 1)] = corr;
    }
}