#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void xcorr(float *d_i1, float *d_i2, float *d_icorr, int m1, int n1, int m2, int n2) {
    // Thread indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Calculate global row and column indices
    int row_out = blockIdx.y * blockDim.y + ty;
    int col_out = blockIdx.x * blockDim.x + tx;

    // Starting points for convolution
    int row_st = row_out - (m2 - 1);
    int col_st = col_out - (n2 - 1);

    // Initialize correlation variable
    float corr = 0.0f;

    // Ensure that the output index is within bounds
    if (row_out >= 0 && row_out < m1 + m2 - 1 && col_out >= 0 && col_out < n1 + n2 - 1) {
        // Optimize memory access by collapsing loops into a single statement
        for (int i = 0; i < m2; i++) { 
            for (int j = 0; j < n2; j++) {
                int img1_row = row_st + i;
                int img1_col = col_st + j;
                if (img1_row >= 0 && img1_row < m1 && img1_col >= 0 && img1_col < n1) {
                    // Access and multiply only when within bounds to save branch instructions
                    corr += d_i1[img1_row + img1_col * m1] * d_i2[i + j * m2];
                }
            }
        }
        // Assign the computed correlation value to the output array
        d_icorr[row_out + col_out * (m1 + m2 - 1)] = corr;
    }
}