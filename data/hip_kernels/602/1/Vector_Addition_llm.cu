#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Vector_Addition(const int *dev_a, const int *dev_b, int *dev_c) {
    // Calculate global thread ID uniquely
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Utilize the stride pattern to allow coalesced memory accesses
    int stride = blockDim.x * gridDim.x;
    
    // Optimize loop with stride and ensure it only processes available work
    for (int i = tid; i < N; i += stride) {
        dev_c[i] = dev_a[i] + dev_b[i];
    }
}