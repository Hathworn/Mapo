#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void expandPlanes_kernel_Fp32_NCHW(float* output, const uint64_t* masks, const float* values, int n) {
    // Block size of 256, same mask/val for 64 consecutive threads.
    constexpr int kNumShmemElements = 256 / 64;

    // Use shared memory to reduce global memory access.
    __shared__ uint64_t shMasks[kNumShmemElements];
    __shared__ float shVals[kNumShmemElements];

    int index = threadIdx.x + blockDim.x * blockIdx.x;

    int planeIndex = index >> 6;

    if (planeIndex >= n) return;

    // Load inputs to shared memory with coalesced access pattern.
    if (threadIdx.x < kNumShmemElements) {
        shMasks[threadIdx.x] = masks[planeIndex + threadIdx.x];
        shVals[threadIdx.x] = values[planeIndex + threadIdx.x];
    }
    __syncthreads();

    // Directly calculate the mask and value using optimized operations.
    uint64_t mask = shMasks[planeIndex % kNumShmemElements];

    // Calculate local index within the block.
    int sqIndex = index & 0x3F;
    float op = 0;

    // Check if the bit is set and load corresponding value.
    if (mask & (1ull << sqIndex)) {
        op = shVals[planeIndex % kNumShmemElements];
    }
    output[index] = op;
}