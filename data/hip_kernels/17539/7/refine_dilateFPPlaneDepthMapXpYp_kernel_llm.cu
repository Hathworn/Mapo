#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refine_dilateFPPlaneDepthMapXpYp_kernel(float* fpPlaneDepthMap, int fpPlaneDepthMap_p, float* maskMap, int maskMap_p, int width, int height, int xp, int yp, float fpPlaneDepth)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Early exit for threads outside of the bounds
    if((x >= width) || (y >= height)) return;

    // Compute target position with offset
    int target_x = x + xp;
    int target_y = y + yp;

    // Check target position bounds condition
    if((target_x >= 0) && (target_y >= 0) && (target_x < width) && (target_y < height))
    {
        float depth = maskMap[y * maskMap_p + x];

        // Update map depth only when depth condition met
        if(depth > 0.0f)
        {
            fpPlaneDepthMap[target_y * fpPlaneDepthMap_p + target_x] = fpPlaneDepth;
        }
    }
}