#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generateGaussian_kernel(float* og, float delta, int radius)
{
    const int x = threadIdx.x - radius;
    const float inv_delta_squared = 1.0f / (2 * delta * delta); // Precompute inverse of delta squared
    og[threadIdx.x] = __expf(-x * x * inv_delta_squared); // Use precomputed value
}