#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float refineDepthSubPixel(const float3& depths, const float3& sims)
{
    float outDepth = -1.0f;

    float simM1 = (sims.x + 1.0f) * 0.5f; // Normalize similarity values to range [0, 1]
    float simP1 = (sims.z + 1.0f) * 0.5f;
    float sim1 = (sims.y + 1.0f) * 0.5f;

    if((simM1 > sim1) && (simP1 > sim1))
    {
        float dispStep = -((simP1 - simM1) / (2.0f * (simP1 + simM1 - 2.0f * sim1)));

        float b = (depths.z + depths.x) * 0.5f; // Calculate midpoint for linear interpolation
        float a = b - depths.x; // Determine slope

        outDepth = a * dispStep + b; // Compute refined depth
    }

    return outDepth;
}

__global__ void refine_computeBestDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float3* isims, int isims_p, float3* idpts, int idpts_p, int width, int height, float simThr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if((x < width) && (y < height))
    {
        float3 depths = idpts[y * idpts_p + x];
        float3 sims = isims[y * isims_p + x];

        // Select best depth and similarity
        float outDepth = ((sims.x < sims.y) ? depths.x : depths.y);
        float outSim = ((sims.x < sims.y) ? sims.x : sims.y);
        outDepth = ((sims.z < outSim) ? depths.z : outDepth);
        outSim = ((sims.z < outSim) ? sims.z : outSim);

        float refinedDepth = refineDepthSubPixel(depths, sims);
        if(refinedDepth > 0.0f)
        {
            outDepth = refinedDepth; // Update outDepth if refinement is valid
        }

        osim[y * osim_p + x] = (outSim < simThr ? outSim : 1.0f);
        odpt[y * odpt_p + x] = (outSim < simThr ? outDepth : -1.0f);
    }
}