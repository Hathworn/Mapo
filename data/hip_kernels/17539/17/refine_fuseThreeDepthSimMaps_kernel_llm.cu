#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refine_fuseThreeDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float* isimLst, int isimLst_p, float* idptLst, int idptLst_p, float* isimAct, int isimAct_p, float* idptAct, int idptAct_p, int width, int height, float simThr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        // Load data for current pixel
        float dpts[3];
        float sims[3];

        dpts[0] = odpt[y * odpt_p + x];
        dpts[1] = idptLst[y * idptLst_p + x];
        dpts[2] = idptAct[y * idptAct_p + x];

        sims[0] = osim[y * odpt_p + x];
        sims[1] = isimLst[y * isimLst_p + x];
        sims[2] = isimAct[y * isimAct_p + x];

        // Find the minimum similarity and corresponding depth
        float outDpt = dpts[0];
        float outSim = sims[0];

        #pragma unroll  // Unroll the loop to improve performance
        for (int i = 1; i < 3; ++i) {
            if (sims[i] < outSim){
                outSim = sims[i];
                outDpt = dpts[i];
            }
        }

        // Write the results back
        osim[y * osim_p + x] = outSim;
        odpt[y * odpt_p + x] = outDpt;
    }
}