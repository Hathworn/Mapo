#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 16  // Define TILE_SIZE if not already defined

__global__ void matrixMultKernel_tiled(float* Ad, float* Bd, float* Cd, int n)
{
    __shared__ float A_shared[TILE_SIZE][TILE_SIZE];
    __shared__ float B_shared[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int column = blockIdx.x * TILE_SIZE + threadIdx.x;

    // Early exit for threads outside matrix bounds
    if (row >= n || column >= n) return;

    float Celem = 0.0;

    // Loop over tiles
    for (int m = 0; m < n / TILE_SIZE; m++)
    {
        // Load tiles into shared memory
        A_shared[threadIdx.y][threadIdx.x] = Ad[row * n + m * TILE_SIZE + threadIdx.x];
        B_shared[threadIdx.y][threadIdx.x] = Bd[(m * TILE_SIZE + threadIdx.y) * n + column];

        __syncthreads(); // Synchronize to make sure the tiles are loaded

        // Compute C element
        for (int k = 0; k < TILE_SIZE; k++)
        {
            Celem += A_shared[threadIdx.y][k] * B_shared[k][threadIdx.x];
        }

        __syncthreads(); // Synchronize before loading new tiles
    }

    // Store the result
    Cd[row * n + column] = Celem;
}