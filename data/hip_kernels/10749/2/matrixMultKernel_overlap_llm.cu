#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMultKernel_overlap(float* Ad, float* Bd, float* Cd, int n)
{
    __shared__ float A_shared[TILE_SIZE][TILE_SIZE];
    __shared__ float B_shared[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int column = blockIdx.x * TILE_SIZE + threadIdx.x;

    // Exit if out of bounds
    if (row >= n || column >= n) {
        return;
    }

    float Celem = 0.0;
    float reg_1 = (row < n && threadIdx.x < TILE_SIZE) ? Ad[row * n + threadIdx.x] : 0.0f;
    float reg_2 = (threadIdx.y < TILE_SIZE && column < n) ? Bd[threadIdx.y * n + column] : 0.0f;

    for (int m = 0; m < (n + TILE_SIZE - 1) / TILE_SIZE; m++) {
        A_shared[threadIdx.y][threadIdx.x] = reg_1;
        B_shared[threadIdx.y][threadIdx.x] = reg_2;

        __syncthreads();

        if (m * TILE_SIZE + threadIdx.x < n && row < n) reg_1 = Ad[row * n + m * TILE_SIZE + threadIdx.x];
        if (m * TILE_SIZE + threadIdx.y < n && column < n) reg_2 = Bd[(m * TILE_SIZE + threadIdx.y) * n + column];

        for (int k = 0; k < TILE_SIZE; k++) {
            Celem += A_shared[threadIdx.y][k] * B_shared[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < n && column < n) {
        Cd[row * n + column] = Celem;
    }
}