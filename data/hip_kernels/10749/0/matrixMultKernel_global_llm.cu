#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMultKernel_global(float* Ad, float* Bd, float* Cd, int n)
{
    // Calculate row and column indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int column = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundary conditions to prevent unnecessary computation
    if(row >= n || column >= n) return;

    float Celem = 0.0;

    // Use a single pointer arithmetic expression to improve readability
    for(int j = 0; j < n; j++) {
        Celem += Ad[row * n + j] * Bd[j * n + column];
    }

    // Store computed value in result matrix
    Cd[row * n + column] = Celem;
}