#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZE 32

__global__ void AddVectors(int* a, int* b, int* c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread calculates its own index
    if (tid < SIZE)
    {
        c[tid] = a[tid] + b[tid];
    }
}