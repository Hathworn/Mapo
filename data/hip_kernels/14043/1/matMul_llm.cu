#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matMul(float *A, int l_A, int c_A, float *B, int l_B, int c_B, float *C, int l_C, int c_C)
{
    float resultat = 0.0;
    int ligne = blockIdx.x * blockDim.x + threadIdx.x;
    int colonne = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread is within matrix bounds
    if(ligne < l_A && colonne < c_B) 
    {
        // Compute matrix multiplication
        for(int i = 0; i < c_A; i++)
        {
            resultat += A[ligne * c_A + i] * B[i * c_B + colonne];
        }
        C[ligne * c_C + colonne] = resultat;
    }
}