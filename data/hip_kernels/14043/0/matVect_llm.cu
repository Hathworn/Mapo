#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Rayhana ZIARA
* produit matrice vecteur
*/

/*
* DESCRIPTION : kernel concerning matrix-vector product
* PARAMETERS : matrix A, vector v, vector r, and size of vectors
* RETURN : /
*/

/*
* DESCRIPTION : function for displaying matrix and vector
* PARAMETERS : matrix to display, number of rows and columns of A,
* RETURN : /
*/
__global__ void matVect(float *A, float *v, float *r, int size)
{
    float resultat = 0.0;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit for out-of-bounds index
    if(index >= size)
    {
        return;
    }

    // Optimized matrix-vector multiplication
    for(int i = 0; i < size; ++i)
    {
        resultat += A[index * size + i] * v[i];
    }

    // Store result in output vector
    r[index] = resultat;
}