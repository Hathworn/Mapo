#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index is within array bounds
    if (idx < numEls) {
        dest[idx] = a[idx] + b[idx];
    }
}