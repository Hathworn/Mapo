#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GetScaleFactorsKernel(float *indata, float *base, float *stdev, float *factors, int nchans, int processed) {

    // Cache variables for better performance
    __shared__ float shared_mean;
    __shared__ float shared_estd;

    // Initialize shared memory
    if (threadIdx.x == 0) {
        shared_mean = 0.0f;
        shared_estd = stdev[threadIdx.x] * stdev[threadIdx.x] * (processed - 1.0f);
    }
    __syncthreads();

    float mean = shared_mean;
    float estd = shared_estd;
    float oldmean = base[threadIdx.x];

    float val = 0.0f;
    float diff = 0.0f;

    // Updated to use local registers as much as possible
    for (int isamp = 0; isamp < 2 * NACCUMULATE; ++isamp) {
        val = indata[isamp * nchans + threadIdx.x];
        diff = val - oldmean;
        mean = oldmean + diff * factors[processed + isamp + 1];
        estd += diff * (val - mean);
        oldmean = mean;
    }

    // Store results back to global memory
    if (threadIdx.x == 0) {
        base[threadIdx.x] = mean;
        stdev[threadIdx.x] = sqrtf(estd / (float)(processed + 2 * NACCUMULATE - 1.0f));
    }
}