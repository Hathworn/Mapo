#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sigmoid64(double* A, int size)
{
    // Optimize index calculation using inbuilt function
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds
    if (idx < size) {
        // Efficient computation of sigmoid function
        A[idx] = 1.0 / (1.0 + exp(-A[idx]));
    }
}