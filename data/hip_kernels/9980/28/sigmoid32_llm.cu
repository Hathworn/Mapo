#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sigmoid32(float* A, int size)
{
    // Combine grid and block calculations for better readability and potential optimization
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 
              (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x + 
              (blockIdx.z * blockDim.z + threadIdx.z) * gridDim.x * gridDim.y * blockDim.x * blockDim.y;

    if (idx < size) {
        // Use expf directly for better performance
        A[idx] = 1.0f / (1.0f + expf(-A[idx]));
    }
}