#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    // Cache grid dimensions and block dimensions for efficiency
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary check
    if (i >= max_x || j >= max_y) return;

    // Cache pixel index calculation for efficiency
    int pixel_index = j * max_x + i;

    // Initialize random state with a fixed seed and unique sequence number
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}