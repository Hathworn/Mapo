#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ int is_between(float a, float b, float c) {
#if between_method == 0
    return (b > a) != (c > a);
#elif between_method == 1
    return ((b <= a) && (c > a)) || ((b > a) && (c <= a));
#elif between_method == 2
    return ((a - b) == 0.0f) || ((a - b) * (a - c) < 0.0f);
#elif between_method == 3
    return ((a - b) * (a - c) < 0.0f) || (a - b == 0.0f);
#endif
}

__global__ void cn_pnpoly(int* bitmap, float2* points, int n) {
    int i = blockIdx.x * block_size_x * tile_size + threadIdx.x;
    if (i >= n) return; // Avoid unnecessary computation

    __shared__ int shared_c[tile_size * block_size_x];  // Use shared memory
    int* c = &shared_c[threadIdx.x * tile_size];

    float2 lpoints[tile_size];
#pragma unroll
    for (int ti = 0; ti < tile_size; ti++) {
        c[ti] = 0;
        if (i + block_size_x * ti < n) {
            lpoints[ti] = points[i + block_size_x * ti];
        }
    }

    int k = VERTICES - 1;
    for (int j = 0; j < VERTICES; k = j++) {
        float2 vj = d_vertices[j];
        float2 vk = d_vertices[k];

#if use_precomputed_slopes == 0
        float slope = (vk.x - vj.x) / (vk.y - vj.y);
#elif use_precomputed_slopes == 1
        float slope = d_slopes[j];
#endif

#pragma unroll
        for (int ti = 0; ti < tile_size; ti++) {
            float2 p = lpoints[ti];

#if use_method == 0
            c[ti] ^= (is_between(p.y, vj.y, vk.y) && (p.x < slope * (p.y - vj.y) + vj.x));
#elif use_method == 1
            int b = is_between(p.y, vj.y, vk.y);
            c[ti] += b & (p.x < vj.x + slope * (p.y - vj.y));
#endif
        }
    }

#pragma unroll
    for (int ti = 0; ti < tile_size; ti++) {
        if (i + block_size_x * ti < n) {
#if use_method == 0
            bitmap[i + block_size_x * ti] = c[ti];
#elif use_method == 1
            bitmap[i + block_size_x * ti] = c[ti] & 1;
#endif
        }
    }
}