#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmul_kernel(float *C, float *A, float *B) {
    // Declare shared memory for sub-matrices of A and B
    __shared__ float sA[block_size_y][block_size_x];
    __shared__ float sB[block_size_y][block_size_x];

    // Define thread index within the block
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    // Define global index for C matrix
    int x = blockIdx.x * block_size_x + tx;
    int y = blockIdx.y * block_size_y + ty;

    // Initialize sum
    float sum = 0.0;

    // Loop over sub-matrices
    for (int k = 0; k < WIDTH; k += block_size_x) {
        // Load sub-matrices from global to shared memory
        sA[ty][tx] = A[y * WIDTH + k + tx];
        sB[ty][tx] = B[(k + ty) * WIDTH + x];
        __syncthreads(); // Ensure all threads have loaded their elements

        // Compute partial product and accumulate
        for (int kb = 0; kb < block_size_x; kb++) {
            sum += sA[ty][kb] * sB[kb][tx];
        }
        __syncthreads(); // Ensure computation is done before next iteration
    }

    // Write result to global memory
    C[y * WIDTH + x] = sum;
}