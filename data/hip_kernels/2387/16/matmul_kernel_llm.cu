#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matmul_kernel(float *C, float *A, float *B) {

    __shared__ float sA[block_size_y][block_size_x * tile_size_y];
    __shared__ float sB[block_size_y][block_size_x * tile_size_x];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = blockIdx.x * block_size_x * tile_size_x + tx;
    int y = blockIdx.y * block_size_y + ty;

    float sum[tile_size_y][tile_size_x] = {0}; // Initialize accumulators

    for (int k = 0; k < WIDTH; k += block_size_x) {

        __syncthreads();
        // Coalesced loading
        #pragma unroll
        for (int i = 0; i < tile_size_y; i++) {
            int y_index = y + i * block_size_y;
            sA[ty][tx + i * block_size_x] = A[y_index * WIDTH + k + tx];
        }

        #pragma unroll
        for (int j = 0; j < tile_size_x; j++) {
            int b_index = (k + ty) * WIDTH + x + j * block_size_x;
            sB[ty][tx + j * block_size_x] = B[b_index];
        }

        __syncthreads();

        // Compute partial results
        #pragma unroll
        for (int kb = 0; kb < block_size_x; kb++) {
            #pragma unroll
            for (int i = 0; i < tile_size_y; i++) {
                float regA = sA[ty][kb + i * block_size_x];
                #pragma unroll
                for (int j = 0; j < tile_size_x; j++) {
                    sum[i][j] += regA * sB[kb][tx + j * block_size_x];
                }
            }
        }
    }

    // Store the final results
    #pragma unroll
    for (int i = 0; i < tile_size_y; i++) {
        int y_index = (y + i * block_size_y) * WIDTH;
        #pragma unroll
        for (int j = 0; j < tile_size_x; j++) {
            C[y_index + x + j * block_size_x] = sum[i][j];
        }
    }
}