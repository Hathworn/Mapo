#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmul_kernel(float *C, float *A, float *B) {

    __shared__ float sA[block_size_y * tile_size_y][block_size_x];
    __shared__ float sB[block_size_x][block_size_x * tile_size_x];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = blockIdx.x * block_size_x * tile_size_x + tx;
    int y = blockIdx.y * block_size_y * tile_size_y + ty;
    int k, kb;

    float sum[tile_size_y][tile_size_x] = {0.0f};

    for (k = 0; k < WIDTH; k += block_size_x) {

        // Load data to shared memory
        #pragma unroll
        for (int i = 0; i < tile_size_y; i++) {
            int sharedY = ty + block_size_y * i;
            sA[sharedY][tx] = A[(y + sharedY - ty) * WIDTH + k + tx];

            #pragma unroll
            for (int j = 0; j < tile_size_x; j++) {
                sB[tx][sharedY * tile_size_x + j] = B[(k + blockIdx.y * block_size_y * tile_size_y + sharedY) * WIDTH + x + j * block_size_x];
            }
        }
        __syncthreads();

        // Matrix multiplication using shared memory
        #pragma unroll
        for (kb = 0; kb < block_size_x; kb++) {

            #pragma unroll
            for (int i = 0; i < tile_size_y; i++) {
                #pragma unroll
                for (int j = 0; j < tile_size_x; j++) {
                    sum[i][j] += sA[ty + block_size_y * i][kb] * sB[kb][tx + j * block_size_x];
                }
            }
        }
        __syncthreads();
    }

    // Store result back to global memory
    #pragma unroll
    for (int i = 0; i < tile_size_y; i++) {
        #pragma unroll
        for (int j = 0; j < tile_size_x; j++) {
            C[y * WIDTH + x + block_size_y * i * WIDTH + j * block_size_x] = sum[i][j];
        }
    }
}