#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cn_pnpoly_naive(int* bitmap, float2* points, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        int c = 0;
        float2 p = points[i];

        int k = VERTICES - 1;

        #pragma unroll // Unroll the loop for better ILP and warp execution efficiency
        for (int j = 0; j < VERTICES; k = j++) {
            float2 vj = d_vertices[j];
            float2 vk = d_vertices[k];

            float slope = (vk.x - vj.x) / (vk.y - vj.y);

            // Using condition short-circuit property, reducing branch divergence
            if ((vj.y > p.y) != (vk.y > p.y) && 
                (p.x < slope * (p.y - vj.y) + vj.x)) {
                c = !c;
            }
        }

        bitmap[i] = c;
    }
}