#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmul_kernel(float *C, float *A, float *B) {
    // Calculate unique index for thread
    int x = blockIdx.x * block_size_x + threadIdx.x;
    int y = blockIdx.y * block_size_y + threadIdx.y;

    // Use shared memory to optimize memory access
    __shared__ float sharedA[block_size_y][WIDTH];
    __shared__ float sharedB[WIDTH][block_size_x];

    // Initialize accumulation variable
    float sum = 0.0;

    // Loop through segments of A and B
    for (int k = 0; k < WIDTH/block_size_y; k++) {
        // Load elements into shared memory
        sharedA[threadIdx.y][k*block_size_y+threadIdx.x] = A[y*WIDTH + k*block_size_y + threadIdx.x];
        sharedB[k*block_size_y+threadIdx.y][threadIdx.x] = B[(k*block_size_y+threadIdx.y)*WIDTH + x];
        __syncthreads(); // Synchronize threads within a block

        // Perform partial dot product using shared memory
        for (int n = 0; n < block_size_y; n++) {
            sum += sharedA[threadIdx.y][n] * sharedB[n][threadIdx.x];
        }
        __syncthreads(); // Synchronize before new segment is loaded
    }

    // Store result in global memory
    C[y*WIDTH + x] = sum;
}