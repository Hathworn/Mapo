#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define domain_width    4096
#define domain_height   2048

__global__ void stencil_kernel(float *x_new, float *x_old) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim for cleaner indexing
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y > 0 && y < domain_height - 1 && x > 0 && x < domain_width - 1) {
        int center = y * domain_width + x; // Precompute center index
        x_new[center] = (
            x_old[center] +
            x_old[center - 1] +
            x_old[center + 1] +
            x_old[center + domain_width] +
            x_old[center - domain_width]
        ) / 5.0f;
    }
}