#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void squareKernel(float* d_in, float* d_out, int threads_num) {
    const unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x; // global id
    if (gid < threads_num) {
        float val = d_in[gid];
        float result = val / (val - 2.3f);
        d_out[gid] = result * result * result; // Use repeated multiplication instead of powf
    }
}