#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int listLength = 700;
__global__ void squareKernel(float* d_in, float *d_out, int threads_num) {
    const unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x; // global id
    if (gid < threads_num) {
        float temp = d_in[gid] / (d_in[gid] - 2.3f); // avoid redundant computation
        d_out[gid] = temp * temp * temp; // use multiplication instead of powf for efficiency
    }
}