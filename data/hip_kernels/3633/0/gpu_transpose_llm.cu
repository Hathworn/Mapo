#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
int row = 0;
int col = 0;
using namespace std;

__global__ void gpu_transpose(float *dst, float *A, int col, int row) {
    // Calculate row and column indices directly
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure indices are within bounds
    if (x < col && y < row) {
        // Swap the elements
        dst[y * col + x] = A[x * row + y];
    }
}