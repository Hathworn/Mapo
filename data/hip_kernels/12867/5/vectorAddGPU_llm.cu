#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAddGPU(float * __restrict__ a, float * __restrict__ b, float * __restrict__ c, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        // Utilize registers for repeated access
        float tmp_a = a[idx];
        float tmp_b = b[idx];
        c[idx] = tmp_a + tmp_b;
    }
}