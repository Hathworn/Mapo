#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define VERTICES 600

__constant__ float2 d_vertices[VERTICES];
__constant__ float d_slopes[VERTICES];

#ifndef block_size_x
#define block_size_x 256
#endif
#ifndef block_size_y
#define block_size_y 1
#endif
#ifndef block_size_z
#define block_size_z 1
#endif

#ifndef tile_size
#define tile_size 1
#endif

__global__ void cn_pnpoly_naive(int* bitmap, float2* points, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        int c = 0;
        float2 p = points[i];

        // Use register to store previous vertex
        float2 vk = d_vertices[VERTICES - 1];

        for (int j = 0; j < VERTICES; j++) {
            float2 vj = d_vertices[j];

            // Precompute comparisons to reduce calculation
            bool condition1 = (vj.y > p.y) != (vk.y > p.y);
            bool condition2 = (p.x < ((vk.x - vj.x) / (vk.y - vj.y)) * (p.y - vj.y) + vj.x);

            if (condition1 && condition2) {
                c = !c;
            }

            // Move to the next vertex
            vk = vj;
        }

        bitmap[i] = c; // 0 if even (out), and 1 if odd (in)
    }
}