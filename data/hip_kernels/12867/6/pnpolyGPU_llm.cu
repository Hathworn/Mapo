#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pnpolyGPU(const float *vertex, float testx, float testy, int* results)
{
    int id = blockIdx.x;

    // Calculate indices using local variables to reduce redundant calculations
    int indexOrigin = (blockIdx.x + 1) * 3;
    int indexDestino = blockIdx.x * 3;

    float vertexOriginX = vertex[indexOrigin];
    float vertexOriginY = vertex[indexOrigin + 1];
    float vertexDestinoX = vertex[indexDestino];
    float vertexDestinoY = vertex[indexDestino + 1];

    // Utilize logical operators efficiently
    bool condition = ((vertexOriginY > testy) != (vertexDestinoY > testy)) &&
                     (testx < (vertexDestinoX - vertexOriginX) * 
                     (testy - vertexOriginY) / (vertexDestinoY - vertexOriginY) + vertexOriginX);
    
    results[id] = condition ? 1 : 0;
}