#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pnpoly_cnGPU(char *cs, const float *px, const float *py, const float *vx, const float *vy, int npoint, int nvert)
{
    extern __shared__ float shared_mem[]; // Use shared memory for vertices only
    float *tvx = shared_mem;
    float *tvy = &shared_mem[nvert];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load vertices into shared memory only once, by the first thread
    if (threadIdx.x < nvert) {
        tvx[threadIdx.x] = vx[threadIdx.x];
        tvy[threadIdx.x] = vy[threadIdx.x];
    }

    __syncthreads();

    if (i < npoint) {
        int j, k, c = 0;
        
        for (j = 0, k = nvert - 1; j < nvert; k = j++) {
            if (((tvy[j] > py[i]) != (tvy[k] > py[i])) && 
                (px[i] < (tvx[k] - tvx[j]) * (py[i] - tvy[j]) / (tvy[k] - tvy[j]) + tvx[j])) 
            {
                c = !c;
            }
        }
        cs[i] = c & 1;
    }
}