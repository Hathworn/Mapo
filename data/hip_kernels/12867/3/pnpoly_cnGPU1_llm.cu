#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pnpoly_cnGPU1(const float *px, const float *py, const float *vx, const float *vy, char* cs, int npoint, int nvert)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < npoint) {
        int j, k;
        char c = 0; // Use char instead of int for 'c' to save space
        float pyi = py[i], pxi = px[i]; // Cache repeated memory loads
        for (j = 0, k = nvert - 1; j < nvert; k = j++) {
            // Simplified condition checking
            bool cond = ((vy[j] > pyi) != (vy[k] > pyi)) && 
                        (pxi < (vx[k] - vx[j]) * (pyi - vy[j]) / (vy[k] - vy[j]) + vx[j]);
            if (cond) c = !c;
        }
        cs[i] = c;
    }
}