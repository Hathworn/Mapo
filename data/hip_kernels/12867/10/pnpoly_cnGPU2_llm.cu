#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pnpoly_cnGPU2(const float *px, const float *py, const float *vx, const float *vy, char* cs, int npoint, int nvert)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use local variables instead of shared memory for per-thread data
    if (i < npoint) {
        float tpx = px[i];
        float tpy = py[i];
        int c = 0;
        
        // Loop unrolling may not be applied here directly due to variable nvert.
        for (int j = 0, k = nvert - 1; j < nvert; k = j++) {
            // Simplify condition by removing redundant calculations
            bool condition1 = (vy[j] > tpy) != (vy[k] > tpy);
            bool condition2 = tpx < (vx[k] - vx[j]) * (tpy - vy[j]) / (vy[k] - vy[j]) + vx[j];
            if (condition1 && condition2)
                c = !c;
        }
        // Store the result
        cs[i] = c & 1;
    }
}