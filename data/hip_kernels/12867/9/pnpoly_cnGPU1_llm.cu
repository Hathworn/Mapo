#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pnpoly_cnGPU1(const float *px, const float *py, const float *vx, const float *vy, char* cs, int npoint, int nvert)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < npoint) {
        int c = 0;
        // Use registers to store index calculations
        float px_i = px[i];
        float py_i = py[i];

        for (int j = 0, k = nvert - 1; j < nvert; k = j++) {
            // Simplified conditional logic for computation efficiency
            bool cond1 = (vy[j] > py_i) != (vy[k] > py_i);
            float vx_diff = vx[k] - vx[j];
            float vy_diff = vy[k] - vy[j];
            bool cond2 = px_i < (vx_diff * (py_i - vy[j]) / vy_diff + vx[j]);
            
            if (cond1 && cond2) {
                c = !c;
            }
        }
        
        cs[i] = c & 1;
    }
}