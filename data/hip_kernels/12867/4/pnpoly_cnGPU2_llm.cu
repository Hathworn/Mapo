#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pnpoly_cnGPU2(const float *px, const float *py, const float *vx, const float *vy, char* cs, int npoint, int nvert)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < npoint) {
        float tpx = px[i]; // Use register variable for tpx
        float tpy = py[i]; // Use register variable for tpy
        int c = 0;

        // Unroll loop for better performance
        #pragma unroll 
        for (int j = 0, k = nvert - 1; j < nvert; k = j++) {
            bool cond1 = (vy[j] > tpy) != (vy[k] > tpy);
            bool cond2 = tpx < (vx[k] - vx[j]) * (tpy - vy[j]) / (vy[k] - vy[j]) + vx[j];
            if (cond1 && cond2)
                c = !c;
        }

        cs[i] = c & 1;
    }
}