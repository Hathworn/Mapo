#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pnpoly_cnGPU(const float *px, const float *py, const float *vx, const float *vy, char* cs, int npoint, int nvert)
{
    extern __shared__ float sharedMemory[]; // Use dynamic shared memory
    float *tvx = sharedMemory;
    float *tvy = sharedMemory + nvert;

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    for (int j = threadIdx.x; j < nvert; j += blockDim.x) {
        tvx[j] = vx[j];
        tvy[j] = vy[j];
    }
    __syncthreads(); // Ensure all vertices are loaded before proceeding

    if (i < npoint) {
        int j, k, c = 0;
        for (j = 0, k = nvert - 1; j < nvert; k = j++) {
            if (((tvy[j] > py[i]) != (tvy[k] > py[i])) &&
                (px[i] < (tvx[k] - tvx[j]) * (py[i] - tvy[j]) / (tvy[k] - tvy[j]) + tvx[j]))
                c = !c;
        }
        cs[i] = c & 1;
    }
}