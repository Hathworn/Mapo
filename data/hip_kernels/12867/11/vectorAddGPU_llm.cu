#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
    // Calculate thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process each item assigned to this thread
    for (int i = idx; i < N; i += blockDim.x * gridDim.x) {
        c[i] = a[i] + b[i];
    }
}