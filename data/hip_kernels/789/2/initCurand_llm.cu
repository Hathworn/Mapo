#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initCurand(hiprandState *state, unsigned long seed, int n_rows) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if (x < n_rows) {
        // Unrolling loop for better performance
        #pragma unroll
        for (int i = 0; i < 1; i++) {
            hiprand_init(seed, x, 0, &state[x]);
        }
    }
}