#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initCurand(hiprandState *state, unsigned long seed, int n_rows){
int x = blockDim.x * blockIdx.x + threadIdx.x;
if(x < n_rows) {
hiprand_init(seed, x, 0, &state[x]);
}
}