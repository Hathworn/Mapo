#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Use built-in variables to simplify index calculation
    int i = hipBlockIdx_y * hipBlockDim_y + hipThreadIdx_y;
    int j = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;

    // Ensure threads outside the bounds do not execute further
    if (i < h && j < w) {
        // Calculate the linear index once and directly set the value
        image[i * w + j] = value;
    }
}