#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    // pixel containing bottom left corner
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    // target pixel integer coords
    int tx = (int) px;
    int ty = (int) py;
    float value = src[image_row_offset + j];
    float weight;

    // Cache the image stride multiplication
    int img_stride_tx_ty = ty * image_stride + tx;

    // Fill the pixel containing bottom right corner
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        weight = dx * dy;
        _atomicAdd(dst + img_stride_tx_ty, value * weight);
        _atomicAdd(normalization_factor + img_stride_tx_ty, weight);
    }

    // Fill the pixel containing bottom left corner
    tx -= 1;
    img_stride_tx_ty -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        weight = (1.0f - dx) * dy;
        _atomicAdd(dst + img_stride_tx_ty, value * weight);
        _atomicAdd(normalization_factor + img_stride_tx_ty, weight);
    }

    // Fill the pixel containing upper left corner
    ty -= 1;
    img_stride_tx_ty -= image_stride;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        weight = (1.0f - dx) * (1.0f - dy);
        _atomicAdd(dst + img_stride_tx_ty, value * weight);
        _atomicAdd(normalization_factor + img_stride_tx_ty, weight);
    }

    // Fill the pixel containing upper right corner
    tx += 1;
    img_stride_tx_ty += 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        weight = dx * (1.0f - dy);
        _atomicAdd(dst + img_stride_tx_ty, value * weight);
        _atomicAdd(normalization_factor + img_stride_tx_ty, weight);
    }
}