#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Compute the thread's unique position in the image
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Return early if the thread is out of bounds
    if (i >= h || j >= w) return;

    // Calculate position in the flattened image array
    const int pos = i * s + j;

    // Retrieve normalization factor and compute inverse scale safely
    float scale = normalization_factor[pos];
    float invScale = __fdividef(1.0f, scale + (scale == 0.0f));

    // Perform normalization by scaling the image value
    image[pos] *= invScale;
}