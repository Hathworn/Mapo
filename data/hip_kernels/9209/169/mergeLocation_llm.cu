#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];

        // Combine memory load operations to improve coalescing
        const float loc_x = loc.x * scale;
        const float loc_y = loc.y * scale;

        // Write results back to global memory
        x[ptidx] = loc_x;
        y[ptidx] = loc_y;
    }
}