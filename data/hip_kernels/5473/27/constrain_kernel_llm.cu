#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index for the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use striding to allow threads to process multiple elements
    for (; i < N; i += gridDim.x * blockDim.x) {
        // Clamp X[i*INCX] to the range [-ALPHA, ALPHA]
        X[i*INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i*INCX]));
    }
}