#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function with thread synchronization
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation for 1D grid
    if (index < N) // Only proceed if index is within bounds
    {
        float mhat = m[index] / (1.f - __powf(B1, t)); // Use intrinsics for faster computation
        float vhat = v[index] / (1.f - __powf(B2, t)); 

        // Update x using computed mhat and vhat
        x[index] = x[index] + rate * mhat / (sqrtf(vhat) + eps);
    }
}