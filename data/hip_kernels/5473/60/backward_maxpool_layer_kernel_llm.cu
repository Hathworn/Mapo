#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    // Calculate height, width, and area
    int h = (in_h + 2*pad) / stride;
    int w = (in_w + 2*pad) / stride;
    int area = (size - 1) / stride;

    // Compute the unique thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return; // Ensure within bounds

    // Precompute stride offsets
    int w_offset = -pad;
    int h_offset = -pad;

    // Derive indices from id
    int index = id;
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    float d = 0.0f;
    
    // Unroll loops for better efficiency
    #pragma unroll
    for(int l = -area; l <= area; ++l){
        int out_h = (i - h_offset) / stride + l;
        if(out_h < 0 || out_h >= h) continue; // Skip invalid entries
        #pragma unroll
        for(int m = -area; m <= area; ++m){
            int out_w = (j - w_offset) / stride + m;
            if(out_w < 0 || out_w >= w) continue; // Skip invalid entries
            
            // Calculate index for delta and validity check
            int out_index = out_w + w * (out_h + h * (k + c * b));
            if(indexes[out_index] == index) {
                d += delta[out_index];
            }
        }
    }
    prev_delta[index] += d; // Accumulate result
}