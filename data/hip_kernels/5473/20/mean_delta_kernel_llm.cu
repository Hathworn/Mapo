#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    // Calculate the global thread index
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Use shared memory to improve memory access efficiency
    extern __shared__ float shared_delta[];
    float sum = 0;

    for (int j = 0; j < batch; ++j) {
        for (int k = threadIdx.x; k < spatial; k += blockDim.x) {
            int index = j * filters * spatial + i * spatial + k;
            sum += delta[index];
        }
    }

    // Synchronize shared memory
    __syncthreads();

    // Reduce sum from shared memory
    atomicAdd(mean_delta + i, sum);

    // Normalize the result
    mean_delta[i] *= (-1.f / sqrtf(variance[i] + .00001f));
}