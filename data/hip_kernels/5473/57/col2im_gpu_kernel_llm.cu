#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void col2im_gpu_kernel(const int n, const float* data_col, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_im) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop for better performance
    int step = blockDim.x * gridDim.x;
    
    while (index < n) {
        float val = 0;
        int w = index % width + pad;
        int h = (index / width) % height + pad;
        int c = index / (width * height);

        // Precompute some common expressions for efficiency
        int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col;
        int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
        int coeff_w_col = (1 - stride * height_col * width_col);

        // Compute start and end for both dimensions using max to ensure non-negative start
        int w_col_start = max((w < ksize) ? 0 : (w - ksize) / stride + 1, 0);
        int w_col_end = min(w / stride + 1, width_col);
        int h_col_start = max((h < ksize) ? 0 : (h - ksize) / stride + 1, 0);
        int h_col_end = min(h / stride + 1, height_col);

        // Unroll the innermost loop to increase throughput
        #pragma unroll
        for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
            #pragma unroll
            for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
                val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
            }
        }
        data_im[index] += val;
        
        index += step; // move to the next index in increments of step
    }
}