#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void make_extra_network_input_kernel(float* dev_x_coors_for_sub, float* dev_y_coors_for_sub, float* dev_num_points_per_pillar, float* dev_x_coors_for_sub_shaped, float* dev_y_coors_for_sub_shaped, float* dev_pillar_feature_mask, const int MAX_NUM_POINTS_PER_PILLAR)
{
    int ith_pillar = blockIdx.x;
    int ith_point = threadIdx.x;
    int ind = ith_pillar * MAX_NUM_POINTS_PER_PILLAR + ith_point;
    
    // Directly assign x and y to shaped arrays
    dev_x_coors_for_sub_shaped[ind] = dev_x_coors_for_sub[ith_pillar];
    dev_y_coors_for_sub_shaped[ind] = dev_y_coors_for_sub[ith_pillar];
    
    // Use conditional operator for mask assignment
    dev_pillar_feature_mask[ind] = ith_point < dev_num_points_per_pillar[ith_pillar] ? 1.0f : 0.0f;
}