#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan_y(int *g_odata, int *g_idata, int n)
{
    extern __shared__ int temp[];
    int thid = threadIdx.x;
    int bid = blockIdx.x;
    int bdim = blockDim.x;
    int gdim = gridDim.x;
    int offset = 1;
    
    // Coalesced memory access
    int ai = 2 * thid;
    int bi = ai + 1;
    int g_ai = bid + ai * gdim;
    int g_bi = g_ai + gdim;
    temp[ai] = g_idata[g_ai];
    temp[bi] = g_idata[g_bi];
    
    for (int d = n >> 1; d > 0; d >>= 1)
    {
        __syncthreads();
        if (thid < d)
        {
            ai = offset * (2 * thid + 1) - 1;
            bi = offset * (2 * thid + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (thid == 0) { temp[n - 1] = 0; }
    
    for (int d = 1; d < n; d *= 2)
    {
        offset >>= 1;
        __syncthreads();
        if (thid < d)
        {
            ai = offset * (2 * thid + 1) - 1;
            bi = offset * (2 * thid + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    
    __syncthreads();

    g_odata[g_ai] = temp[bi]; // Optimize: Directly writing updated value
    if (bi + 1 == n) {
        g_odata[g_bi] = temp[bi] + g_idata[g_bi];
    } else {
        g_odata[g_bi] = temp[bi + 1];
    }
}