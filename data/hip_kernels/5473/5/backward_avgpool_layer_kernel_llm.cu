#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Get the global threadId
    if(id >= n) return;

    // Calculate indices for batch (b) and channel (k)
    int k = id % c;
    int b = id / c;

    int out_index = k + c * b;
    float out_value = out_delta[out_index] / (w * h);

    // Unroll and optimize the loop with stride to improve performance
    for(int i = 0; i < w * h; i += blockDim.x) {
        int in_index = i + h * w * (k + b * c) + threadIdx.x;
        if (in_index < w * h * c * b) {
            in_delta[in_index] += out_value;
        }
    }
}