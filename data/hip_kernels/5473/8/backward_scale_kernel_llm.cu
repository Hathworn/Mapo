#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Unrolling the loop for better performance
    for(int b = 0; b < batch; ++b){
        for(int i = 0; i < size; i += BLOCK*4){
            int index1 = p + i + size*(filter + n*b);
            int index2 = p + i + BLOCK + size*(filter + n*b);
            int index3 = p + i + 2*BLOCK + size*(filter + n*b);
            int index4 = p + i + 3*BLOCK + size*(filter + n*b);
            sum += (p+i < size) ? delta[index1]*x_norm[index1] : 0;
            sum += (p+i+BLOCK < size) ? delta[index2]*x_norm[index2] : 0;
            sum += (p+i+2*BLOCK < size) ? delta[index3]*x_norm[index3] : 0;
            sum += (p+i+3*BLOCK < size) ? delta[index4]*x_norm[index4] : 0;
        }
    }

    part[p] = sum;
    __syncthreads();
    
    // Use atomic operations to prevent race condition
    if(p == 0) {
        for(int i = 0; i < BLOCK; ++i) atomicAdd(&scale_updates[filter], part[i]);
    }
}