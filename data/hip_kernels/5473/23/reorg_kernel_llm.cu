#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Pre-compute values to reduce repetitive arithmetic operations
    int area = w * h;
    int slice = area * c;
    int out_area = w * stride * h * stride;

    // Extract input dimensions
    int b = i / slice;
    int remainder = i % slice;
    int in_c = remainder / area;
    remainder = remainder % area;
    int in_h = remainder / w;
    int in_w = remainder % w;

    // Compute output dimensions
    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    
    // Calculate the output index
    int b_stride = b * out_area * out_c;
    int out_index = w2 + w * stride * (h2 + h * stride * c2) + b_stride;

    // Write output based on the forward flag
    if (forward)
        out[out_index] = x[in_index];
    else
        out[in_index] = x[out_index];
}