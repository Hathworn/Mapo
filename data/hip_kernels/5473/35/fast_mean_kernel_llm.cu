#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    // Unroll loop to enhance performance
    for(j = 0; j < batch; ++j) {
        for(i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += x[index];
        }
    }
    
    __syncthreads();

    // Use a parallel reduction for summation
    for (int stride = threads / 2; stride > 0; stride >>= 1) {
        if (id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    if (id == 0) {
        mean[filter] = local[0] / (spatial * batch);
    }
}