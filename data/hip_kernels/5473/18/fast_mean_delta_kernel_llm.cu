#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;
    int stride = blockDim.x * gridDim.x;  // Use grid-stride loop for parallelization

    for(int j = 0; j < batch; ++j) {
        for(int i = id; i < spatial; i += stride) {  // Use grid-stride loop
            int index = j*spatial*filters + filter*spatial + i;
            local[id] += delta[index];
        }
    }

    __syncthreads();

    // Use reduction to sum values in shared memory
    for(int stride = threads / 2; stride > 0; stride /= 2) {
        if(id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    // Store the result in global memory from the first thread
    if(id == 0) {
        mean_delta[filter] = local[0] * (-1.f/sqrtf(variance[filter] + .00001f));
    }
}