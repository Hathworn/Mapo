#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Utilize faster memory: register
    float bias_value = biases[filter];

    if(offset < size) 
    {
        // Optimize memory access pattern
        int index = (batch * n + filter) * size + offset;
        output[index] *= bias_value;
    }
}