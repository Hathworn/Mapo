#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *__restrict__ pred, float *__restrict__ truth, float *__restrict__ delta, float *__restrict__ error)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    // Ensure the thread is within bounds
    if (i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);

        // Use conditional operator to simplify branching
        error[i] = (abs_val < 1) ? diff * diff : 2 * abs_val - 1;
        delta[i] = (abs_val < 1) ? diff : (diff > 0 ? 1 : -1);
    }
}