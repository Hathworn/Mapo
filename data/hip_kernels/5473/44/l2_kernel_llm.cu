#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation
    if (i < n) {
        float diff = truth[i] - pred[i];
        float diff_squared = diff * diff;
        error[i] = diff_squared; // Calculating squared difference
        delta[i] = diff; // Assign the difference to delta
    }
}