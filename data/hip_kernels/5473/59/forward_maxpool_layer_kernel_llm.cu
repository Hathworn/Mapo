#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    // Calculate the output dimensions
    int h = (in_h + 2*pad) / stride;
    int w = (in_w + 2*pad) / stride;
    int c = in_c;

    // Compute the unique global index for each thread
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    // Compute the mapping from flat index to 5D indices
    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    int w_offset = -pad;
    int h_offset = -pad;

    // Initialize the maximum value to negative infinity and index to -1
    int out_index = j + w*(i + h*(k + c*b));
    float max = -INFINITY;
    int max_i = -1;

    // Loop over the pooling window to find the maximum value
    for(int l = 0; l < size; ++l) {
        for(int m = 0; m < size; ++m){
            int cur_h = h_offset + i*stride + l;
            int cur_w = w_offset + j*stride + m;
            int index = cur_w + in_w*(cur_h + in_h*(k + b*in_c));
            // Check if current position is valid within input dimensions
            bool valid = (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w);
            float val = valid ? input[index] : -INFINITY;
            // Update max value and index when a new maximum is found
            if (val > max) {
                max = val;
                max_i = index;
            }
        }
    }
    
    // Store the maximum value and its index in the output and indexes arrays
    output[out_index] = max;
    indexes[out_index] = max_i;
}