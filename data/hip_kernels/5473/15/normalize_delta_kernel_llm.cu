#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // For processing beyond the block

    for (int i = index; i < N; i += stride) { // Loop to cover all elements
        int f = (i / spatial) % filters;
        
        float inv_sqrt_var = 1.f / sqrtf(variance[f] + .00001f); // Pre-calculate inverse sqrt variance
        float val_delta = variance_delta[f] * 2.f * (x[i] - mean[f]) / (spatial * batch) + mean_delta[f] / (spatial * batch);
        
        delta[i] = delta[i] * inv_sqrt_var + val_delta; 
    }
}