#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use effective tensor core access pattern if possible
    for (; i < N; i += blockDim.x * gridDim.x) {
        // Perform the operation
        Y[i * INCY] = powf(X[i * INCX], ALPHA);
    }
}