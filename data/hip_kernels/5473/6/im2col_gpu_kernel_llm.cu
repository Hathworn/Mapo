#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void im2col_gpu_kernel(const int n, const float* data_im, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_col) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    while (index < n) {
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;

        // Pointer calculations
        float* data_col_ptr = data_col + (channel_out * height_col + h_out) * width_col + w_out;
        const float *data_im_ptr = data_im + (channel_in * height + h_in) * width + w_in;

        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                // Ensure the indices are within bounds
                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ? data_im_ptr[i * width + j] : 0;

                // Move to the next position in the output column
                data_col_ptr += height_col * width_col;
            }
        }
        // Move to the next index
        index += blockDim.x * gridDim.x;
    }
}