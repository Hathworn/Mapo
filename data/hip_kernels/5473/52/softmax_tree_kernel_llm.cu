#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;

    // Compute the largest element
    for(i = 0; i < n; ++i){
        float val = input[i*stride];
        largest = fmaxf(val, largest);
    }

    // Compute the exponential values and accumulate their sum
    for(i = 0; i < n; ++i){
        float e = expf((input[i*stride] - largest) / temp);
        sum += e;
        output[i*stride] = e;
    }

    // Normalize the output by the sum of exponentials
    float inv_sum = 1.0f / sum; // Use reciprocal for division optimization
    for(i = 0; i < n; ++i){
        output[i*stride] *= inv_sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    // Flatten grid and block indexing to 1D
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int total_elements = spatial * batch * groups;

    // Return if beyond bounds
    if (id >= total_elements) return;

    // Decompose 1D ID to s, g, b indices
    int s = id % spatial;
    id /= spatial;
    int g = id % groups;
    int b = id / groups;

    // Compute data offsets
    int goff = group_offset[g] * spatial;
    int boff = b * stride;

    // Call device function with computed offsets and parameters
    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}