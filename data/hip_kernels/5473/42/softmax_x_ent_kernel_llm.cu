#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to compute softmax cross-entropy loss
__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if(i < n){
        float t = truth[i];
        float p = pred[i];

        // Use FMA for better precision
        error[i] = (t > 0) ? -__logf(p) : 0;
        
        // Direct subtraction, no need to check truth value again
        delta[i] = t - p;
    }
}