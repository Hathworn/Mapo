#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify thread index calculation
    if (i >= filters) return;
    
    float sum = 0; // Use a local variable to accumulate the sum
    for (int j = 0; j < batch; ++j) {
        int base_index = j * filters * spatial + i * spatial; // Calculate base index once per batch
        for (int k = 0; k < spatial; ++k) {
            sum += x[base_index + k]; // Index calculation outside innermost loop
        }
    }
    mean[i] = sum * scale;
}