#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Improved boundary check to avoid unnecessary calculations
    if(i * INCX < N) {
        X[i * INCX] = ALPHA;
    }
}