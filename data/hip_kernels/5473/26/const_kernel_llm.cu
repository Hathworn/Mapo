#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the unique global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure grid coverage before access
    if (i < N) {
        X[i * INCX] = ALPHA;
    }
}