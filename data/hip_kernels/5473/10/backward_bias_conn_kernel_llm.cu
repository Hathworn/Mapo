#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation
    if (index >= n) return;
    float sum = 0.0f;
    for (int b = 0; b < batch; ++b) {
        sum += delta[b * n + index]; // Unroll the index calculation inside the loop
    }
    atomicAdd(&bias_updates[index], sum); // Use atomicAdd to ensure thread safety
}