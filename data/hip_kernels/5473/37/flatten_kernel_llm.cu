#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= N) return; // Ensure within bounds

    // Precompute divisors to reduce repeated calculations
    int div_spatial = spatial;
    int div_layers = layers * spatial;

    // Calculate indices using precomputed divisors
    int in_s = i % div_spatial;
    int in_c = (i / div_spatial) % layers;
    int b = i / div_layers;

    // Simplify memory access expressions
    int i1 = b * div_layers + in_c * spatial + in_s;
    int i2 = b * div_layers + in_s * layers + in_c;

    // Use conditional assignment with forward flag
    out[forward ? i2 : i1] = x[forward ? i1 : i2];
}