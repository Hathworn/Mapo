#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread processes valid data and avoid unused threads
    for (; i < n; i += blockDim.x * gridDim.x) {
        c[i] += a[i] * b[i];
    }
}