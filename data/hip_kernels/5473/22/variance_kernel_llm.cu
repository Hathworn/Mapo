#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Scale factor to normalize variance
    float scale = 1.f / (batch * spatial - 1);
    float sum = 0; // Local variable for accumulation

    // Prefetch mean value
    float mean_val = mean[i];

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            float diff = x[index] - mean_val; // Subtract mean
            sum += diff * diff; // Accumulate squared difference
        }
    }

    variance[i] = sum * scale; // Scale the accumulated sum
}