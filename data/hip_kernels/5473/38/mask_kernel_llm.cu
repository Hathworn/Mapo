#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify block index calculation
    int stride = gridDim.x * blockDim.x;          // Define stride for processing large arrays

    for(; i < n; i += stride) {                    // Process elements with stride to maximize GPU utilization
        if(mask[i] == mask_num) {                  // Use more readable code block structure
            x[i] = val;
        }
    }
}