#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the thread's global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop to process larger arrays
    int stride = gridDim.x * blockDim.x;
    
    // Ensure all threads work on valid elements only
    for (; i < N; i += stride) {
        X[i * INCX] += ALPHA;
    }
}