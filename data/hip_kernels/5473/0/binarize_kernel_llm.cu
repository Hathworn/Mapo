#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(const float * __restrict__ x, const int n, float * __restrict__ binary)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        // Use ternary operation for binarization
        binary[i] = (x[i] >= 0.0f) ? 1.0f : -1.0f;
    }
}