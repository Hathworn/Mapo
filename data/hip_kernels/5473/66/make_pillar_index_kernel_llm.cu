#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void make_pillar_index_kernel(int* dev_pillar_count_histo, int* dev_counter, int* dev_pillar_count, 
                                         int* dev_x_coors, int* dev_y_coors, 
                                         float* dev_x_coors_for_sub, float* dev_y_coors_for_sub, 
                                         float* dev_num_points_per_pillar, int* dev_sparse_pillar_map, 
                                         const int max_pillars, const int max_points_per_pillar, 
                                         const int GRID_X_SIZE, const float PILLAR_X_SIZE, const float PILLAR_Y_SIZE, 
                                         const int NUM_INDS_FOR_SCAN) {
    int x = blockIdx.x;
    int y = threadIdx.x;
    int num_points_at_this_pillar = dev_pillar_count_histo[y * GRID_X_SIZE + x];

    // Exit early if no points at this pillar
    if (num_points_at_this_pillar == 0) {
        return;
    }

    int count = atomicAdd(dev_counter, 1);

    // Ensure count is within max_pillars
    if (count < max_pillars) {
        atomicAdd(dev_pillar_count, 1);

        // Clamp number of points per pillar to max_points_per_pillar
        dev_num_points_per_pillar[count] = min(num_points_at_this_pillar, max_points_per_pillar);

        dev_x_coors[count] = x;
        dev_y_coors[count] = y;

        // Calculate coordinates for sub
        dev_x_coors_for_sub[count] = x * PILLAR_X_SIZE + 0.1f;
        dev_y_coors_for_sub[count] = y * PILLAR_Y_SIZE - 39.9f;

        // Update sparse pillar map
        dev_sparse_pillar_map[y * NUM_INDS_FOR_SCAN + x] = 1;
    }
}