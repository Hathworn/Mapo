#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial) {
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;

    int b = index / spatial;
    int i = index % spatial;
    int offset = b * filters * spatial + i;
    float sum = 0.0f;

    // Unroll the loop for better performance
    #pragma unroll
    for (int f = 0; f < filters; ++f) {
        int idx = offset + f * spatial;
        sum += x[idx] * x[idx];
    }

    sum = sqrtf(sum);
    if (sum == 0.0f) sum = 1.0f;

    // Combine operations to reduce memory accesses
    for (int f = 0; f < filters; ++f) {
        int idx = offset + f * spatial;
        float x_val = x[idx] / sum;
        x[idx] = x_val;
        dx[idx] = (1.0f - x_val) / sum;
    }
}