#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        // Precalculate to minimize redundant operations
        int f = (index / spatial) % filters;
        float inv_std = 1.0f / sqrtf(variance[f] + 0.00001f);
        x[index] = (x[index] - mean[f]) * inv_std;
    }
}