#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Use shared memory to reduce data access latency
    __shared__ float shared_weights[1024]; // Adjust size based on block size
    int num_iter = (size + blockDim.x - 1) / blockDim.x;

    float mean = 0.0f;
    for (int i = 0; i < num_iter; ++i) {
        int index = f * size + i * blockDim.x + threadIdx.x;
        if (index < n * size) {
            shared_weights[threadIdx.x] = fabsf(weights[index]);
        } else {
            shared_weights[threadIdx.x] = 0.0f;
        }
        __syncthreads();

        // Accumulate mean for current block
        for (int j = 0; j < blockDim.x; ++j) {
            if (index < n * size) {
                mean += shared_weights[j];
            }
        }
        __syncthreads();
    }

    mean /= size;

    for (int i = 0; i < num_iter; ++i) {
        int index = f * size + i * blockDim.x + threadIdx.x;
        if (index < n * size) {
            binary[index] = (weights[index] > 0) ? mean : -mean;
        }
    }
}