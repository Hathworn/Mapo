#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    local[id] = 0.0f;  // Initialize shared memory

    int i, j;
    for (j = 0; j < batch; ++j) {
        for (i = id; i < spatial; i += threads) {  // Ensure all threads participate
            int index = j * spatial * filters + filter * spatial + i;

            if (i < spatial)  // Perform bounds check before computation
                local[id] += delta[index] * (x[index] - mean[filter]);
        }
    }
    __syncthreads();

    if (id == 0) {
        float sum = 0.0f;
        for (i = 0; i < threads; ++i) {
            sum += local[i];  // Use local sum to reduce global update frequency
        }
        variance_delta[filter] = sum * -.5f * powf(variance[filter] + .00001f, -1.5f);  // Simplified powf computation
    }
}