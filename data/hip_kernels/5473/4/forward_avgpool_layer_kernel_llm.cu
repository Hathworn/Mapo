#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    int b = id / c; // Simplified division calculation

    float sum = 0.0f; // Accumulate sum locally
    int area = w * h; // Pre-compute area to avoid repetitive calculation
    int out_index = k + c * b;
    int base_index = h * w * (k + b * c); // Pre-compute base index

    for(int i = 0; i < area; ++i) {
        sum += input[i + base_index]; // Use pre-computed base index
    }
    
    output[out_index] = sum / area; // Directly assign averaged value to output
}