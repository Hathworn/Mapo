#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate the global thread index.
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid stride loop to allow handling of more elements beyond a single grid's capacity.
    for (; i < n; i += blockDim.x * gridDim.x) {
        // Compute weighted sum.
        c[i] = s[i] * a[i] + (1.0f - s[i]) * (b ? b[i] : 0.0f);
    }
}