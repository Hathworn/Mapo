#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified thread index calculation
    int stride = gridDim.x * blockDim.x;  // Define stride for grid stride loop

    for (; i < N; i += stride) {  // Use grid stride loop for better performance
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
    }
}