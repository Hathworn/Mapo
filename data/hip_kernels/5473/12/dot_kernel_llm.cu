#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;
    
    // Use precomputed indexes outside the inner loop for better performance
    for (int b = 0; b < batch; ++b) {
        int base_index1 = b * size * n + f1 * size;
        int base_index2 = b * size * n + f2 * size;
        for (int i = 0; i < size; ++i) {
            int i1 = base_index1 + i;
            int i2 = base_index2 + i;
            float val1 = output[i1];
            float val2 = output[i2];

            sum += val1 * val2;
            norm1 += val1 * val1;
            norm2 += val2 * val2;
        }
    }
    
    norm1 = sqrtf(norm1); // Use sqrtf for better precision with float
    norm2 = sqrtf(norm2);
    float norm = norm1 * norm2;
    sum = sum / norm;
    
    for (int b = 0; b < batch; ++b) {
        int base_index1 = b * size * n + f1 * size;
        int base_index2 = b * size * n + f2 * size;
        for (int i = 0; i < size; ++i) {
            int i1 = base_index1 + i;
            int i2 = base_index2 + i;
            delta[i1] += -scale * sum * output[i2] / norm;
            delta[i2] += -scale * sum * output[i1] / norm;
        }
    }
}