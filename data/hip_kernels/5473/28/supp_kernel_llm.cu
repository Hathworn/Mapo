#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // Calculate unique thread ID
    int gridStride = gridDim.x * blockDim.x; // Calculate stride for grid stride loop
    float alphaSquare = ALPHA * ALPHA; // Pre-compute ALPHA squared
    
    for (int i = tid; i < N; i += gridStride) {
        float val = X[i * INCX];
        if (val * val < alphaSquare) {
            X[i * INCX] = 0;
        }
    }
}