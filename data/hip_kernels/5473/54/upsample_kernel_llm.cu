#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation
    if (i >= N) return;

    int b = i / (w * h * c * stride * stride);  // Calculate batch
    i = i % (w * h * c * stride * stride);
    
    int out_c = (i / (w * stride * h * stride)) % c;  // Calculate output channel
    int out_h = (i / (w * stride)) % (h * stride);  // Calculate output height
    int out_w = i % (w * stride);  // Calculate output width
    
    int in_w = out_w / stride;
    int in_h = out_h / stride;

    int in_index = b*w*h*c + out_c*w*h + in_h*w + in_w;  // Optimized for clarity

    if (forward) {
        out[i] += scale * x[in_index];  // Forward computation simplified
    } else {
        atomicAdd(&x[in_index], scale * out[i]);  // Use pointer directly
    }
}