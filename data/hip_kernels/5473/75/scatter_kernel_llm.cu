#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scatter_kernel(int *x_coors, int *y_coors, float *pfe_output, float *scattered_feature, const int MAX_NUM_PILLARS_, const int GRID_X_SIZE, const int GRID_Y_SIZE)
{
    int i_pillar = blockIdx.x;
    int num_features = blockDim.x;

    // Use shared memory for thread synchronization
    __shared__ int x_shared;
    __shared__ int y_shared;

    if (threadIdx.x == 0) {
        x_shared = x_coors[i_pillar];
        y_shared = y_coors[i_pillar];
    }

    // Synchronize threads to ensure shared memory is set
    __syncthreads();

    // Load feature data and update output
    if (i_pillar < MAX_NUM_PILLARS_ && threadIdx.x < num_features) {
        int idx = threadIdx.x * MAX_NUM_PILLARS_ + i_pillar;
        float feature = pfe_output[idx];
        int output_idx = threadIdx.x * GRID_Y_SIZE * GRID_X_SIZE + y_shared * GRID_X_SIZE + x_shared;
        scattered_feature[output_idx] = feature;
    }
}