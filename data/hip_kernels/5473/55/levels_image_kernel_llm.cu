#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Device function for converting HSV to RGB
__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    float h = hsv.x;
    float s = hsv.y;
    float v = hsv.z;

    float r, g, b;
    float f, p, q, t;

    if (s == 0) {
        r = g = b = v;
    } else {
        int index = (int)floorf(h);
        f = h - index;
        p = v * (1 - s);
        q = v * (1 - s * f);
        t = v * (1 - s * (1 - f));
        switch (index) {
            case 0: r = v; g = t; b = p; break;
            case 1: r = q; g = v; b = p; break;
            case 2: r = p; g = v; b = t; break;
            case 3: r = p; g = q; b = v; break;
            case 4: r = t; g = p; b = v; break;
            default: r = v; g = p; b = q; break;
        }
    }
    r = fminf(fmaxf(r, 0), 1);
    g = fminf(fmaxf(g, 0), 1);
    b = fminf(fmaxf(b, 0), 1);
    return make_float3(r, g, b);
}

// Device function for converting RGB to HSV
__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    float r = rgb.x;
    float g = rgb.y;
    float b = rgb.z;

    float h, s, v;
    float max = fmaxf(fmaxf(r, g), b);
    float min = fminf(fminf(r, g), b);
    float delta = max - min;

    v = max;
    if (max != 0) {
        s = delta / max;
        if (r == max) {
            h = (g - b) / delta;
        } else if (g == max) {
            h = 2 + (b - r) / delta;
        } else {
            h = 4 + (r - g) / delta;
        }
        h = (h < 0) ? h + 6 : h;
    } else {
        s = 0;
        h = -1;
    }
    return make_float3(h, s, v);
}

// Optimized CUDA kernel for image processing
__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int size = batch * w * h;
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Optimized index computation
    if (id >= size) return;

    float rshift = rand[0];
    float gshift = rand[1];
    float bshift = rand[2];
    float r0 = rand[8 * id + 0];
    float r1 = rand[8 * id + 1];
    float r2 = rand[8 * id + 2];
    float r3 = rand[8 * id + 3];

    saturation = r0 * (saturation - 1) + 1;
    saturation = (r1 > .5f) ? 1.f / saturation : saturation;
    exposure = r2 * (exposure - 1) + 1;
    exposure = (r3 > .5f) ? 1.f / exposure : exposure;

    size_t offset = id * w * h * 3;  // Corrected offset calculation
    image += offset;
    int idx_r = id % w + w * (id / w % h);
    int idx_g = idx_r + w * h;
    int idx_b = idx_g + w * h;

    float r = image[idx_r];
    float g = image[idx_g];
    float b = image[idx_b];
    float3 rgb = make_float3(r, g, b);

    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }
    image[idx_r] = rgb.x * scale + translate + (rshift - .5f) * shift;
    image[idx_g] = rgb.y * scale + translate + (gshift - .5f) * shift;
    image[idx_b] = rgb.z * scale + translate + (bshift - .5f) * shift;
}