#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit early if index exceeds groups
    if (i >= groups) return;

    float tmp_sum = 0.0f; // Local variable to reduce global memory write
    for (int k = 0; k < n; ++k) {
        tmp_sum += x[k * groups + i];
    }
    sum[i] = tmp_sum;
}