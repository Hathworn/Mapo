#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate global thread index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll loops for increased performance
    #pragma unroll
    for (; i < N; i += blockDim.x * gridDim.x) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}