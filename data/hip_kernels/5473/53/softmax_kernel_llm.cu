#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    float sum = 0;
    float largest = -INFINITY;

    // Calculate maximum value
    for (int i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = (val > largest) ? val : largest;
    }

    // Compute exponentials and sum
    for (int i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp); // Optimize calculation using largest
        sum += e;
        output[i * stride] = e;
    }

    // Normalize the output values
    float invSum = 1.0f / sum; // Calculate multiplicative inverse once
    for (int i = 0; i < n; ++i) {
        output[i * stride] *= invSum; // Use multiplication instead of division for better performance
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= batch * groups) return;
    
    int b = id / groups;
    int g = id % groups;

    softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}