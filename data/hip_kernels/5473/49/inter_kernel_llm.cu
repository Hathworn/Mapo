#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D grid for simplicity
    int totalThreads = gridDim.x * blockDim.x; // Total number of threads
    int totalElements = (NX + NY) * B; // Total number of elements to process

    for (int idx = i; idx < totalElements; idx += totalThreads) { // Strided loop for workload distribution
        int b = idx / (NX + NY);
        int j = idx % (NX + NY);

        if (j < NX) {
            OUT[idx] = X[b * NX + j]; // Assign from X array
        } else {
            OUT[idx] = Y[b * NY + j - NX]; // Assign from Y array
        }
    }
}