#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCK 256 // Assumed BLOCK size, usually better defined globally

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;

    // Reduce `part` array initialization and accumulation
    float sum = 0.0f;

    // Unrolling loop for better performance
    for(int b = 0; b < batch; ++b){
        for(int i = 0; i < size; i += BLOCK){
            int index = p + i + size * (filter + n * b);
            if (p + i < size) {
                sum += delta[index];
            }
        }
    }

    // Use warp reduction for shared memory accumulation
    part[p] = sum;
    __syncthreads();

    // Optimized reduction with warp
    if (p < 32) { 
        for(int i = p + 32; i < BLOCK; i += 32) {
            sum += part[i];
        }
    }

    // Final bias update with atomic operation
    if (p == 0) {
        atomicAdd(&bias_updates[filter], sum);
    }
}