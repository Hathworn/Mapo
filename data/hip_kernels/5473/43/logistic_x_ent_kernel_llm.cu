#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Use shared memory for improved memory access performance
    extern __shared__ float shared_mem[];
    float *shared_pred = shared_mem;
    float *shared_truth = shared_mem + blockDim.x;
    
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    
    // Load data into shared memory
    if (i < n) {
        shared_pred[threadIdx.x] = pred[i];
        shared_truth[threadIdx.x] = truth[i];
    }
    __syncthreads();
    
    if(i < n){
        float t = shared_truth[threadIdx.x];
        float p = shared_pred[threadIdx.x];
        error[i] = -t * logf(p + 1e-7f) - (1 - t) * logf(1 - p + 1e-7f);
        delta[i] = t - p;
    }
}