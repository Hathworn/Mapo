#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access and reduce potential warp divergence
__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    while (i < n) {
        if (mask[i] == mask_num) {
            x[i] *= scale;
        }
        i += stride;  // Ensure all elements are processed
    }
}