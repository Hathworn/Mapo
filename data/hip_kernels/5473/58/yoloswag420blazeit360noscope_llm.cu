#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    // Calculate the global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;

    // Loop unrolling for better performance in case of large data size
    for (; id < size; id += gridSize) {
        input[id] = (rand[id] < prob) ? 0 : input[id] * scale;
    }
}