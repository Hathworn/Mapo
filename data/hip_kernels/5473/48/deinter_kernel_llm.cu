#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use strided loop to allow better utilization of threads
    for (; i < (NX + NY) * B; i += stride) {
        int b = i / (NX + NY);
        int j = i % (NX + NY);
        
        if (j < NX) {
            if (X) X[b * NX + j] += OUT[i];
        } else {
            if (Y) Y[b * NY + j - NX] += OUT[i];
        }
    }
}