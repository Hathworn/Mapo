#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0.0f;

    // Process elements in a coalesced manner
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {  // Use thread stride loop
            int index = j * spatial * filters + filter * spatial + i;
            float diff = x[index] - mean[filter];
            local[id] += diff * diff;
        }
    }
    
    __syncthreads();

    // Use reduction to sum results within a block
    if(id < 32) {
        for (int offset = threads / 2; offset > 0; offset >>= 1) {
            if(id < offset) {
                local[id] += local[id + offset];
            }
            __syncthreads();  // Ensure all writes to local complete
        }
    }

    // Store final sum into variance
    if (id == 0) {
        variance[filter] = local[0] / (spatial * batch - 1);
    }
}