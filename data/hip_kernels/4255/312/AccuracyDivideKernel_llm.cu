#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use a single thread for division, as only one operation is needed
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}