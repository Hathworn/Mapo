#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c, int n) {
    // Use a more descriptive variable name for index
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll the loop for potentially better performance
    if (globalIdx < n) {
        c[globalIdx] = a[globalIdx] + b[globalIdx];
    }
}