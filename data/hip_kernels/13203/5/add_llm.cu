#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    // Calculate unique thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform boundary check to avoid out-of-bounds access
    if (index < gridDim.x * blockDim.x) {
        c[index] = a[index] + b[index];
    }
}