#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to add elements of two arrays with optimization
__global__ void add(int *a, int *b, int *c) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    c[idx] = a[idx] + b[idx]; // Perform addition using global index
}