#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    int idx = threadIdx.x; // Use thread index for operation
    c[idx] = a[idx] + b[idx]; // Utilize parallel execution for element-wise addition
}