#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matMultCuda(float *cu_C, float *cu_A, float *cu_B, unsigned int n) {
    // Calculate row and column index for current thread
    int row = (blockIdx.x * blockDim.x) + threadIdx.x;
    int col = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Ensure thread is within matrix bounds
    if (row < n && col < n) {
        float temp_sum = 0.0f; // Use float instead of int for precise computation

        // Utilize shared memory for faster data access
        extern __shared__ float shared_A[], shared_B[];
        shared_A[threadIdx.y * n + threadIdx.x] = cu_A[row * n + threadIdx.x];
        shared_B[threadIdx.y * n + threadIdx.x] = cu_B[threadIdx.y * n + col];
        __syncthreads();

        for (int elem = 0; elem < n; elem++) {
            temp_sum += shared_A[threadIdx.y * n + elem] * shared_B[elem * n + col];
        }

        cu_C[row * n + col] = temp_sum; // Store result
    }
}