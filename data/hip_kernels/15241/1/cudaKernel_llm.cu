#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaKernel(int n, double* gpuWeights, int* gpuG, int* gpuTempGrid, int *flag)
{
    // Moment's coordinates in the grid //
    int momentCol = blockIdx.x * blockDim.x + threadIdx.x;
    int momentRow = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Shared memory allocated for weights //
    __shared__ double sharedWeights[25];
    // Shared memory allocated for a block of moments //
    // Size is (BLOCK_SIZE+4)^2 //
    int sharedSize = (BLOCK_SIZE+4);
    __shared__ int sharedG[(BLOCK_SIZE+4)*(BLOCK_SIZE+4)];
    
    // Moment's coordinates in the shared memory //
    int sharedRow = threadIdx.y + 2;
    int sharedCol = threadIdx.x + 2;

    // Store weights in shared memory //
    if (threadIdx.x < 5 && threadIdx.y < 5)
        sharedWeights[threadIdx.x * 5 + threadIdx.y] = gpuWeights[threadIdx.x * 5 + threadIdx.y];

    // Synchronize to ensure sharedWeights are fully loaded //
    __syncthreads();

    // In this double loop, moments and their necessary neighbours are //
    // passed from global to shared memory sequentially //
    for (int i = momentRow; i < n + 2; i += blockDim.y * gridDim.y)
    {
        for (int j = momentCol; j < n + 2; j += blockDim.x * gridDim.x)
        {
            // Store moment in shared memory //
            sharedG[sharedRow * sharedSize + sharedCol] = gpuG[((i + n) % n) * n + ((j + n) % n)];
            
            if (threadIdx.x < 2)
            {
                int idxRow = (i + n) % n;

                // Load boundaries //
                sharedG[sharedRow * sharedSize + sharedCol - 2] = gpuG[idxRow * n + (-2 + j + n) % n];
                sharedG[sharedRow * sharedSize + sharedCol + BLOCK_SIZE] = gpuG[idxRow * n + (BLOCK_SIZE + j + n) % n];

                if (threadIdx.y < 2)
                {
                    // Load corners //
                    sharedG[(sharedRow - 2) * sharedSize + sharedCol - 2] = gpuG[((-2 + i + n) % n) * n + (-2 + j + n) % n];
                    sharedG[(sharedRow + BLOCK_SIZE) * sharedSize + sharedCol - 2] = gpuG[((i + n + BLOCK_SIZE) % n) * n + (-2 + j + n) % n];
                    sharedG[(sharedRow - 2) * sharedSize + sharedCol + BLOCK_SIZE] = gpuG[((-2 + i + n) % n) * n + (j + n + BLOCK_SIZE) % n];
                    sharedG[(sharedRow + BLOCK_SIZE) * sharedSize + sharedCol + BLOCK_SIZE] = gpuG[((i + n + BLOCK_SIZE) % n) * n + (j + n + BLOCK_SIZE) % n];
                }
            }
            
            if (threadIdx.y < 2)
            {
                int idxRowTop = (-2 + i + n) % n;
                int idxRowBottom = (i + n + BLOCK_SIZE) % n;

                // Load top and bottom boundaries //
                sharedG[(sharedRow - 2) * sharedSize + sharedCol] = gpuG[idxRowTop * n + (j + n) % n];
                sharedG[(sharedRow + BLOCK_SIZE) * sharedSize + sharedCol] = gpuG[idxRowBottom * n + (j + n) % n];
            }
            
            __syncthreads();

            // Compute the spins of moments within the grid //
            if (i < n && j < n)
            {
                double weightFactor = 0.0;
                
                for (int row = 0; row < 5; row++)
                {
                    for (int col = 0; col < 5; col++)
                    {
                        if (col == 2 && row == 2) continue;
                        weightFactor += sharedG[(sharedRow - 2 + row) * sharedSize + sharedCol - 2 + col] * sharedWeights[row * 5 + col];
                    }
                }

                // Determine future atomic spin value based on weight factor //
                if (abs(weightFactor) < 0.0001)
                {
                    gpuTempGrid[n * i + j] = sharedG[sharedRow * sharedSize + sharedCol];
                }
                else if (weightFactor > 0.00001)
                {
                    gpuTempGrid[n * i + j] = 1;
                    if (gpuG[n * i + j] == -1)
                    {
                        *flag = 1;
                    }
                }
                else
                {
                    gpuTempGrid[n * i + j] = -1;
                    if (gpuG[n * i + j] == 1)
                    {
                        *flag = 1;
                    }
                }
            }
            __syncthreads();
        }
    }
}