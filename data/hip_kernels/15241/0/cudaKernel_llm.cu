#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void cudaKernel(int n, double* gpuWeights, int* gpuG, int* gpuTempGrid, int *flag)
{
    // Moment's coordinates in the grid
    int momentCol = blockIdx.x * blockDim.x + threadIdx.x;
    int momentRow = blockIdx.y * blockDim.y + threadIdx.y;

    double weightFactor = 0.0;

    // Each thread calculates the spin for a block of moments
    for (int i = momentRow; i < n; i += blockDim.y * gridDim.y)
    {
        for (int j = momentCol; j < n; j += blockDim.x * gridDim.x)
        {
            weightFactor = 0.0;
            // Calculate total neighbourhood influence
            for (int weightsRow = 0; weightsRow < 5; weightsRow++)
            {
                for (int weightsCol = 0; weightsCol < 5; weightsCol++)
                {
                    if (weightsCol == 2 && weightsRow == 2)
                        continue;

                    // Calculate neighbour's coordinates using modulus for boundary conditions
                    int gridRowIdx = (weightsRow - 2 + i + n) % n;
                    int gridColIdx = (weightsCol - 2 + j + n) % n;

                    // Reduce to minimize reads by using shared memory (if applicable)
                    weightFactor += gpuG[gridRowIdx * n + gridColIdx] * gpuWeights[weightsRow * 5 + weightsCol];
                }
            }

            // Update moment's atomic spin
            if (weightFactor < 0.0001 && weightFactor > -0.0001)
            {
                gpuTempGrid[n * i + j] = gpuG[n * i + j];
            }
            else if (weightFactor > 0.00001)
            {
                gpuTempGrid[n * i + j] = 1;
                if (gpuG[n * i + j] == -1)
                {
                    atomicOr(flag, 1); // Atomic operation to avoid race condition
                }
            }
            else
            {
                gpuTempGrid[n * i + j] = -1;
                if (gpuG[n * i + j] == -1)
                {
                    atomicOr(flag, 1); // Atomic operation to avoid race condition
                }
            }
        }
    }
}