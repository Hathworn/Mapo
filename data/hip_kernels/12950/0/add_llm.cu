#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int n, float *a, float *b, float *sum)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop to handle larger arrays
    for (int index = i; index < n; index += blockDim.x * gridDim.x) 
    {
        sum[index] = a[index] + b[index];
    }
}