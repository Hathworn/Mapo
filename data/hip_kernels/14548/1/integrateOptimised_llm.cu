#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define INTERVALS 1000000
#define THREADS 512
#define BLOCKS 64

double calculatePiCPU();

__global__ void integrateOptimised(int *n, float *g_sum)
{
    // Calculate global thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int tx = threadIdx.x;

    // Shared memory to hold the sum for each block
    __shared__ float s_sum[THREADS];

    float sum = 0.0f;
    float step  = 1.0f / (float)(*n);

    // Unroll the loop to improve performance
    for (int i = idx + 1; i <= *n; i += blockDim.x * gridDim.x)
    {
        float x = step * ((float)i - 0.5f);
        sum += 4.0f / (1.0f + x*x);
    }
    s_sum[tx] = sum * step;

    // Synchronize threads within the block
    __syncthreads();

    // Use reduction pattern with fewer synchronization points
    for (int i = blockDim.x / 2; i > 0; i >>= 1)
    {
        if (tx < i)
        {
            // Accumulate partial results into shared memory
            s_sum[tx] += s_sum[tx + i];
        }
        __syncthreads();
    }

    // Write the block's result to global memory, only by one thread
    if (tx == 0) g_sum[blockIdx.x] = s_sum[0];
}