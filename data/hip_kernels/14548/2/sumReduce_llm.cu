#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define INTERVALS 1000000

// Max number of threads per block
#define THREADS 512
#define BLOCKS 64

double calculatePiCPU();

// Synchronous error checking call. Enable with nvcc -DDEBUG
__global__ static void sumReduce(int *n, float *g_sum)
{
    int tx = threadIdx.x;
    __shared__ float s_sum[THREADS];

    // Initialize shared memory
    s_sum[tx] = (tx < BLOCKS) ? g_sum[tx * THREADS] : 0.0f;
    __syncthreads(); // Ensure all shared memory is initialized

    // Use optimized loop for reduction
    for (int i = blockDim.x / 2; i > 0; i >>= 1)
    {
        if (tx < i)
        {
            s_sum[tx] += s_sum[tx + i];
        }
        __syncthreads(); // Synchronize threads before the next iteration
    }

    // Write result from shared memory to global memory
    if (tx == 0) g_sum[blockIdx.x] = s_sum[0];
}