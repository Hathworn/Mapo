#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define INTERVALS 1000000
#define THREADS 512
#define BLOCKS 64

double calculatePiCPU();

// Optimized global kernel function
__global__ void integrateSimple(float *sum, float step, int threads, int blocks)
{
    // Utilizing shared memory for intermediate results to reduce global memory access
    __shared__ float local_sum[THREADS];

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int local_idx = threadIdx.x;

    local_sum[local_idx] = 0.0f;

    for (int i = idx; i < INTERVALS; i += threads * blocks)
    {
        float x = (i + 0.5f) * step;
        local_sum[local_idx] += 4.0f / (1.0f + x*x);
    }

    // Synchronize threads within block
    __syncthreads();

    // Reduce sum within the block using first thread
    if (local_idx == 0) {
        float block_sum = 0.0f;
        for (int i = 0; i < THREADS; ++i) {
            block_sum += local_sum[i];
        }
        atomicAdd(sum, block_sum);
    }
}