#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NodesApplyTramplingEffectKernel(float* target, float* distanceToPath, int graphW, int graphH, float pathThickness, float tramplingCoefficient)
{
    // Calculate global indexes with boundary adjustments
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

    // Optimize boundary check and computation
    if(i <= graphW && j <= graphH) {
        int index = i + j * (graphW + 2);
        float t = distanceToPath[index];

        // Limit 't' within [0, 1] after normalizing with path thickness
        t = max(0.0f, min(1.0f, fabsf(t / pathThickness)));

        // Cubic parabola computation for trampling effect
        t = t * (t * (-4 * t + 6) - 3) + 1;

        // Atomic addition on target element with trampling coefficient
        atomicAdd(&target[index], t * tramplingCoefficient);
    }
}