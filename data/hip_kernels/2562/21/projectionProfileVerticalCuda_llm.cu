#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void projectionProfileVerticalCuda( const uint8_t * image, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * projection )
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Combine bounds check and atomic operation to minimize thread divergence
    if ( x < width ) {
        atomicAdd( &projection[y], image[y * rowSize + x] );
    }
}