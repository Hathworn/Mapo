#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void isAnyEqualCuda( const uint8_t * image, uint8_t * value, size_t valueCount, uint32_t width, uint32_t height, uint32_t * differenceCount )
{
    // Calculate x and y indices
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if indices are within bounds
    if ( x < width && y < height ) {
        const uint32_t id = y * width + x; // Linear index calculation

        bool equal = false;
        
        // Loop unrolling for faster comparison
        #pragma unroll
        for ( uint32_t i = 0; i < valueCount; ++i ) {
            if ( image[id] == value[i] ) {
                equal = true;
                break;
            }
        }

        // Atomic operation to increment differenceCount
        if ( equal )
            atomicAdd( differenceCount, 1 );
    }
}