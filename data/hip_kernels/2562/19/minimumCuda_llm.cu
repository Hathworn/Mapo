#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void minimumCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
    // Calculate global thread index
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure within bounds
    if ( x < width && y < height ) {
        // Simplified pointer arithmetic with fewer operations
        uint32_t offset = y * rowSizeIn1 + x;
        const uint8_t * in1X = in1 + offset;               
        const uint8_t * in2X = in2 + offset;
        uint8_t * outX = out + offset;
        
        // Use of ternary operator for minimum calculation
        (*outX) = (*in1X < *in2X) ? *in1X : *in2X;
    }
}