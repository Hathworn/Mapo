#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyCuda(const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate the unique index for every thread in a 1D manner to improve global memory coalescing
    const uint32_t index = blockIdx.y * blockDim.y * rowSizeOut + blockIdx.x * blockDim.x + threadIdx.y * rowSizeOut + threadIdx.x;
    const uint32_t pixelIndex = blockIdx.y * blockDim.y + threadIdx.y;
    const uint32_t rowElements = blockDim.x * gridDim.x;
    
    if (pixelIndex < height && threadIdx.x < width) {
        out[index] = in[pixelIndex * rowSizeIn + threadIdx.x];
    }
}