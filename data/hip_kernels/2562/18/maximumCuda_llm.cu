#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void maximumCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate pixel coordinate
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Check bounds to avoid out-of-bounds memory access
    if (x < width && y < height) {
        // Directly calculate the index for all arrays
        const uint32_t indexIn1 = y * rowSizeIn1 + x;
        const uint32_t indexIn2 = y * rowSizeIn2 + x;
        const uint32_t indexOut = y * rowSizeOut + x;

        // Use ternary operator to find the maximum value
        out[indexOut] = (in1[indexIn1] > in2[indexIn2]) ? in1[indexIn1] : in2[indexIn2];
    }
}