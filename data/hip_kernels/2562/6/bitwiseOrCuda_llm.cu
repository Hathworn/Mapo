#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitwiseOrCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height) 
{
    // Calculate unique thread index
    const uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t totalThreads = gridDim.x * blockDim.x;

    // Process multiple elements per thread
    for(uint32_t i = tid; i < width * height; i += totalThreads) {
        const uint32_t x = i % width;
        const uint32_t y = i / width;
        
        const uint32_t idIn1 = y * rowSizeIn1 + x;
        const uint32_t idIn2 = y * rowSizeIn2 + x;
        const uint32_t idOut = y * rowSizeOut + x;
        
        out[idOut] = in1[idIn1] | in2[idIn2];
    }
}