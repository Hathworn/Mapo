#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogramCuda( const uint8_t * data, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * histogram )
{
    // Calculate global thread ID only once
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if ( x < width && y < height ) {
        // Direct index calculation using predefined variables
        const uint32_t id = y * rowSize + x;
        
        // Use register variable for data access to reduce memory latency
        uint8_t dataValue = data[id];
        
        // Atomic add operation for histogram update
        atomicAdd( &histogram[dataValue], 1 );
    }
}