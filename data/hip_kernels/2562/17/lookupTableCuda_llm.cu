#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lookupTableCuda(const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height, uint8_t * table)
{
    // Calculating global thread index for 1D behavior, potentially enhances memory coalescing
    const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * rowSizeIn;

    if ((index % rowSizeIn < width) && (index / rowSizeIn < height)) {
        out[index % rowSizeIn + (index / rowSizeIn) * rowSizeOut] = table[in[index]];
    }
}