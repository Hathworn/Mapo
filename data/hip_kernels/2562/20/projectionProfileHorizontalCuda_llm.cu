#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void projectionProfileHorizontalCuda(const uint8_t * image, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * projection) {
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Use shared memory to reduce atomic operations
    __shared__ uint32_t sharedProjection[256]; // Adjust size as necessary (e.g., equal to blockDim.x)
    if (threadIdx.y == 0) {
        sharedProjection[threadIdx.x] = 0;
    }
    __syncthreads();

    if (x < width && y < height) {
        const uint8_t * imageX = image + y * rowSize + x;
        atomicAdd(&sharedProjection[threadIdx.x], (*imageX));
    }
    __syncthreads();

    if (threadIdx.y == 0) {
        atomicAdd(&projection[x], sharedProjection[threadIdx.x]);
    }
}