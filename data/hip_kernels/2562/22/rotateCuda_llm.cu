#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rotateCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, float inXStart, float inYStart, uint32_t width, uint32_t height, float cosAngle, float sinAngle )
{
    uint32_t outX = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t outY = blockDim.y * blockIdx.y + threadIdx.y;

    // Only process valid pixels within the boundary
    if ( outX < width && outY < height ) {
        const float exactInX = inXStart + cosAngle * outX + sinAngle * outY;
        const float exactInY = inYStart - sinAngle * outX + cosAngle * outY;
        const int32_t inX = static_cast<int32_t>(exactInX);
        const int32_t inY = static_cast<int32_t>(exactInY);

        // Compute output pixel address
        uint8_t *outPixel = out + outY * rowSizeOut + outX;

        // Check if input coordinates are within valid range
        if ( inX < 0 || inX >= width - 1 || inY < 0 || inY >= height - 1 ) {
            *outPixel = 0; // Set out-of-bound values to 0
        } else {
            // Compute input pixel address
            const uint8_t *inPixel = in + inY * rowSizeIn + inX;

            // Bilinear interpolation
            const float probX = exactInX - inX;
            const float probY = exactInY - inY;
            const float mean = inPixel[0] * (1 - probX) * (1 - probY) +
                               inPixel[1] * probX * (1 - probY) +
                               inPixel[rowSizeIn] * (1 - probX) * probY +
                               inPixel[rowSizeIn + 1] * probX * probY +
                               0.5f;

            *outPixel = static_cast<uint8_t>(mean);
        }
    }
}