#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitwiseAndCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate global thread index once, use register to minimize recalculation
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        // Precompute row index to reduce redundant computation
        const uint32_t rowIn1 = y * rowSizeIn1;
        const uint32_t rowIn2 = y * rowSizeIn2;
        const uint32_t rowOut = y * rowSizeOut;
        
        // Load and store memory with computed indices
        out[rowOut + x] = in1[rowIn1 + x] & in2[rowIn2 + x];
    }
}