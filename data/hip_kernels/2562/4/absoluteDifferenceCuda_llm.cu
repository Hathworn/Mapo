#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void absoluteDifferenceCuda(const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height) 
{
    // Calculate global indices for x and y
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        // Directly compute offset without using additional pointers
        const uint32_t offsetIn1 = y * rowSizeIn1 + x;
        const uint32_t offsetIn2 = y * rowSizeIn2 + x;
        const uint32_t offsetOut = y * rowSizeOut + x;
        
        // Simplified absolute difference calculation
        out[offsetOut] = abs(in1[offsetIn1] - in2[offsetIn2]);
    }
}