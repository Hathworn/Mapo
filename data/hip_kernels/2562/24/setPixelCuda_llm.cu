#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to set pixel value
__global__ void setPixelCuda(uint8_t *in, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t *pointX, uint32_t *pointY, uint32_t pointSize, uint32_t value) {
    // Calculate unique global thread ID
    const uint32_t idPoint = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform calculations only if the thread ID is within valid range
    if (idPoint < pointSize) {
        // Cache x and y coordinates in registers
        const uint32_t x = __ldg(&pointX[idPoint]);
        const uint32_t y = __ldg(&pointY[idPoint]);

        // Check if the pixel is within the image bounds
        if (x < width && y < height) {
            // Set the pixel value
            in[y * rowSize + x] = value;
        }
    }
}