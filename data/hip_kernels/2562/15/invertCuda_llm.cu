#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void invertCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height) 
{
    // Calculate global thread index for 2D grid
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within image boundaries
    if (x < width && y < height) {
        // Perform bitwise inversion
        out[y * rowSizeOut + x] = ~in[y * rowSizeIn + x];
    }
}