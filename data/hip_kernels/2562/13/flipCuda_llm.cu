#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flipCuda(const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height, bool horizontal, bool vertical)
{
    // Cache block and thread indices
    const uint32_t inX = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t inY = blockDim.y * blockIdx.y + threadIdx.y;

    // Check bounds
    if (inX < width && inY < height) {
        // Calculate output coordinates
        const uint32_t outX = horizontal ? (width - 1 - inX) : inX;
        const uint32_t outY = vertical ? (height - 1 - inY) : inY;

        // Perform memory transfer with calculated indices
        out[outY * rowSizeOut + outX] = in[inY * rowSizeIn + inX];
    }
}
