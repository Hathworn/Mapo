#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitwiseXorCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
    // Use a single variable for 2D to 1D mapping
    const uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;

    // Combine conditionals for bounds checking
    if ( idx < width && idy < height ) {
        const uint32_t idOffsetIn1 = idy * rowSizeIn1 + idx;
        const uint32_t idOffsetIn2 = idy * rowSizeIn2 + idx;
        const uint32_t idOffsetOut = idy * rowSizeOut + idx;
        out[idOffsetOut] = in1[idOffsetIn1] ^ in2[idOffsetIn2];
    }
}