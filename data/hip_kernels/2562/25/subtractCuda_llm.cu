#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtractCuda(const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate global thread indices
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Process only valid pixels within image boundaries
    if (x < width && y < height) {
        // Compute linear index for each input and output
        uint32_t indexIn1 = y * rowSizeIn1 + x;
        uint32_t indexIn2 = y * rowSizeIn2 + x;
        uint32_t indexOut = y * rowSizeOut + x;

        // Compute subtraction and apply threshold
        out[indexOut] = (in1[indexIn1] > in2[indexIn2]) ? (in1[indexIn1] - in2[indexIn2]) : 0;
    }
}