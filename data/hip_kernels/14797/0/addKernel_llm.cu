#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

// Kernel function with improved parallel execution setup
__global__ void addKernel(int *c, const int *a, const int *b, unsigned int size) 
{
    // Calculate global index based on block and thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within the array bounds
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}