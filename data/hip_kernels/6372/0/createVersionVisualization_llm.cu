#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/**
* @file
* @brief     CIS 565 Version Checker
* @details   A simple CUDA hello-world-style program for Patrick Cozzi's
*            CIS 565: GPU Programming, at the University of Pennsylvania.
* @authors   Starter code: Yining Karl Li, Liam Boone, Harmony Li, Kai Ninomiya
* @copyright University of Pennsylvania
*/

__global__ void createVersionVisualization(uchar4* PBOpos, int width, int height, int major, int minor) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Check if the thread is within bounds
    if (x < width && y < height) {
        int index = x + (y * width);

        // Set initial pixel color to black
        PBOpos[index] = {0, 0, 0, 0};

        // Select version based on y position
        int ver = y < height / 2 ? major : minor;

        // Set pixel color based on version
        switch (ver) {
            case 0:
                PBOpos[index].x = 255;
                break;
            case 1:
                PBOpos[index].y = 255;
                break;
            case 2:
                PBOpos[index].z = 255;
                break;
            case 3:
                PBOpos[index].x = 255;
                PBOpos[index].y = 255;
                break;
            case 5:
                PBOpos[index].z = 255;
                PBOpos[index].y = 255;
                break;
        }
    }
}