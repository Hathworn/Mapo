#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* kernel.cu */

__global__ void AddVector(int vecSize, const float* vecA, const float* vecB, float* vecC)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Use bound check to exit early if out of range
    if (i >= vecSize) return;

    vecC[i] = vecA[i] + vecB[i];
}