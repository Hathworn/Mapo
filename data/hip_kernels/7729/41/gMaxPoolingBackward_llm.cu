#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gMaxPoolingBackward(float* adj, int adjRows, int adjCols, float* in, float* adjIn, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Return early if thread ID is out of bounds
    if(tid >= adjRows * adjCols) return;

    int rowId = tid / adjCols; // Corrected row index calculation
    int colId = tid % adjCols; // Corrected column index calculation

    float* b = in + (rowId * inCols) + (colId * width);

    // Adjust width for the last column
    width = (colId == adjCols - 1 ? lastWidth : width);
    
    float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;
    
    // Initialize with zero for clarity
    size_t currentMaxIdx = 0;
    float currentMaxVal = b[0] * localMask[0];
    
    // Unrolled loop for better performance (example with 4-way unrolling, adjust as needed)
    for(int i = 1; i < width; ++i) {
        float val = b[i] * localMask[i];
        if(val > currentMaxVal) {
            currentMaxIdx = i;
            currentMaxVal = val;
        }
    }

    // Atomic operation to avoid race conditions
    atomicAdd(&adjIn[(rowId * inCols) + (colId * width) + currentMaxIdx], adj[rowId + (colId * adjCols)]);
}