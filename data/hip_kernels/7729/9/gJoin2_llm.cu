#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gJoin2(float* out, size_t rowBatch, size_t cols, const float* in1, size_t inStride1, const float* in2, size_t inStride2) {
    const int outStride = inStride1 + inStride2;
    const int rows = rowBatch * outStride;

    // Optimize block-level iteration
    int j = blockIdx.x + blockIdx.y * gridDim.x;
    if (j < rows) {
        float* rowOut = out + j * cols;

        int curBatch = j / outStride;
        int curPos = j % outStride;

        int jIn1 = (curBatch * inStride1) + curPos;
        int jIn2 = (curBatch * inStride2) + curPos - inStride1;

        const float* rowIn1 = in1 + jIn1 * cols;
        const float* rowIn2 = in2 + jIn2 * cols;

        // Optimize thread-level iteration
        int i = threadIdx.x;
        if (i < cols) {
            if (curPos < inStride1)
                rowOut[i] = rowIn1[i];
            else
                rowOut[i] = rowIn2[i];
        }
    }
}