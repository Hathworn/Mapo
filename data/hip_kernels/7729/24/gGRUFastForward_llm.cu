#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableSigmoid(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gGRUFastForward(float* out, const float* state, const float* xW, const float* sU, const float* b, const float* mask, size_t rows, size_t cols, bool final) {
    // Assign unique thread and block IDs
    int j = blockIdx.x;
    int i = threadIdx.x;

    if (j < rows) {
        float m = !mask || mask[j];
        float* rowOut = out + j * cols;
        const float* rowState = state + j * cols;

        const float* xWrow = xW + j * cols * 3;
        const float* sUrow = sU + j * cols * 3;

        if (i < cols) {
            // Compute helper indices for the GRU components
            int k = i + cols;
            int l = i + 2 * cols;

            // Compute gate and candidate activations
            float r = stableSigmoid(xWrow[i] + sUrow[i] + b[i]);
            float z = stableSigmoid(xWrow[k] + sUrow[k] + b[k]);
            
            float h;
            if (final)
                h = tanhf(xWrow[l] + (sUrow[l] + b[l]) * r);
            else
                h = tanhf(xWrow[l] + sUrow[l] * r + b[l]);

            // Compute the output
            float out = (1.0f - z) * h + z * rowState[i];
            rowOut[i] = m * out + (1 - m) * rowState[i];
        }
    }
}