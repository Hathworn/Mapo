#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gLogSoftmaxGrad(float* grad, const float* adj, const float* val, const int rows, const int cols) {
    for(int bid = blockIdx.x; bid < rows; bid += gridDim.x) {  // Use blockIdx.x as loop start
        if(bid < rows) {
            extern __shared__ float _share[];
            float* _sum = _share;
            
            float* gradRow = grad + bid * cols;
            const float* adjRow = adj + bid * cols;
            const float* valRow = val + bid * cols;

            float sum = 0.0;  // Use a local register for sum

            for(int tid = threadIdx.x; tid < cols; tid += blockDim.x) {  // Utilize thread index properly
                sum += adjRow[tid];
            }
            
            // Reduction
            atomicAdd(&_sum[0], sum);
            __syncthreads();  // Ensure reduction completion
            
            // Compute gradient using local sum
            for(int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
                gradRow[tid] += adjRow[tid] - (expf(valRow[tid]) * _sum[0]);
            }
        }
    }
}