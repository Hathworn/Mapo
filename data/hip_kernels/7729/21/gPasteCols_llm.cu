#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gPasteCols(float* out, const float* in, size_t rows, size_t colsOut, const size_t* targetColIdx, size_t colsIn) {
    int j = blockIdx.x + blockIdx.y * gridDim.x; // Calculate global row index
    while (j < rows) { // Iterate over row blocks
        const float* rowIn = in + j * colsIn;
        float* rowOut = out + j * colsOut;
        
        int i = threadIdx.x + blockIdx.z * blockDim.x; // Calculate global column index
        while(i < colsIn) { // Iterate over column blocks
            atomicAdd(&rowOut[targetColIdx[i]], rowIn[i]); // Use atomic operation to prevent race conditions
            i += blockDim.x * gridDim.z; // Move to the next column block
        }
        
        j += gridDim.x * gridDim.y; // Move to the next row block
    }
}