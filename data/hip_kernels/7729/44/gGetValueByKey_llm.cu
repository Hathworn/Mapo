#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gGetValueByKey(float* d_in, float* d_out, int* indeces, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation
    if(tid < n) {
        d_out[tid] = d_in[indeces[tid]]; // Direct access to memory using computed index
    }
}