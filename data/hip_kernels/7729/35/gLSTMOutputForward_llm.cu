#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableSigmoid(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gLSTMOutputForward(float* out, const float* cell, const float* xW, const float* sU, const float* b, size_t rows, size_t cols) {
    int j = blockIdx.x + blockIdx.y * gridDim.x;  // Calculate row index
    if (j < rows) {
        float* rowOut = out + j * cols;
        const float* rowCell = cell + j * cols;
        const float* xWrow = xW + j * cols * 4;
        const float* sUrow = sU + j * cols * 4;

        int i = threadIdx.x;  // Simplify thread indexing
        if (i < cols) {
            int k = i + 3 * cols;  // Efficient index calculation
            float go = stableSigmoid(xWrow[k] + sUrow[k] + b[k]);
            rowOut[i] = go * tanhf(rowCell[i]);
        }
    }
}