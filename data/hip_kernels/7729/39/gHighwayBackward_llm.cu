#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableSigmoid(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gHighwayBackward(float* out1, float* out2, float* outt, const float* in1, const float* in2, const float* t, const float* adj, size_t length) {
    // Calculate the global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop through elements, in case there are more elements than threads
    while(index < length) {
        float sigma = stableSigmoid(t[index]);
        out1[index] = sigma * adj[index];
        out2[index] = (1.f - sigma) * adj[index];
        outt[index] = sigma * (1.f - sigma) * (in1[index] - in2[index]) * adj[index];

        // Move to the next element for this thread
        index += blockDim.x * gridDim.x;
    }
}