#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gCopyRows(float* out, const float* in, size_t cols, const size_t* sourceRowIdx, size_t rows) {
    // Optimize grid-stride loop for better occupancy and coalescing
    for(int j = blockIdx.x + blockIdx.y * gridDim.x; j < rows; j += gridDim.x * gridDim.y) {
        size_t dstId = j;
        size_t srcId = sourceRowIdx[j];

        float* rowOut = out + dstId * cols;
        const float* rowIn = in + srcId * cols;

        // Optimize thread-stride loop for better coverage and efficiency
        for(int i = threadIdx.x; i < cols; i += blockDim.x) {
            rowOut[i] = rowIn[i];
        }
    }
}