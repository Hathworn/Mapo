#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gSetSparse(float* out, const size_t* indices, const float* values, int length) {
    // Calculate the global index once outside the loop
    int index = blockIdx.x * blockDim.x + threadIdx.x; 
    if(index < length) {
        out[indices[index]] = values[index];
    }
    // Use grid-stride loop for better utilization
    for(index += blockDim.x * gridDim.x; index < length; index += blockDim.x * gridDim.x) {
        out[indices[index]] = values[index];
    }
}