#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableSigmoid(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gLSTMCellForward(float* out, const float* cell, const float* xW, const float* sU, const float* b, const float* mask, size_t rows, size_t cols) {
    // Calculate the row index this block is going to work with
    int j = blockIdx.x * blockDim.y + threadIdx.y;
    if(j < rows) {
        float m = !mask || mask[j];
        
        float* rowOut = out + j * cols;
        const float* rowCell = cell + j * cols;

        const float* xWrow = xW + j * cols * 4;
        const float* sUrow = sU + j * cols * 4;

        // Use shared memory to improve memory access efficiency
        extern __shared__ float sharedMemory[];
        float* shared_xW = sharedMemory;
        float* shared_sU = sharedMemory + cols;
        float* shared_b = sharedMemory + 2 * cols;

        for(int i = threadIdx.x; i < cols; i += blockDim.x) {
            shared_b[i] = b[i];
        }
        __syncthreads();

        for(int i = threadIdx.x; i < cols; i += blockDim.x) {
            float gf = stableSigmoid(xWrow[i] + sUrow[i] + shared_b[i]);

            int k = i + cols;
            float gi = stableSigmoid(xWrow[k] + sUrow[k] + shared_b[k]);

            int l = i + 2 * cols;
            float gc = tanhf(xWrow[l] + sUrow[l] + shared_b[l]);

            float cout = gf * rowCell[i] + gi * gc;
            rowOut[i] = m * cout + (1 - m) * rowCell[i];
        }
    }
}