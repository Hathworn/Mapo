#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gSoftmaxGrad(float* grad, const float* adj, const float* val, const int rows, const int cols) {
    // Use shared memory to reduce global memory access
    extern __shared__ float _share[];
    float* _sum = _share + blockDim.x;
    
    int j = blockIdx.x + blockIdx.y * gridDim.x;
    int stride = gridDim.x * gridDim.y;
    
    for (; j < rows; j += stride) {
        float* gradRow = grad + j * cols;
        const float* adjRow = adj + j * cols;
        const float* valRow = val + j * cols;
        _sum[threadIdx.x] = 0.0f;

        // Sum with stride loop to ensure full warp utilization
        for (int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
            _sum[threadIdx.x] += valRow[tid] * adjRow[tid];
        }

        // Reduce sum within the block
        __syncthreads();
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                _sum[threadIdx.x] += _sum[threadIdx.x + s];
            }
            __syncthreads();
        }

        float summedValue = _sum[0];
        
        // Compute the gradient using the computed sum
        for (int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
            float val = valRow[tid] * (adjRow[tid] - summedValue);
            if (val)
                gradRow[tid] += val;
        }
    }
}