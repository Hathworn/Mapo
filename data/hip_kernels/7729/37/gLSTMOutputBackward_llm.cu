#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableSigmoid(float x) {
  if(x >= 0) {
    float z = expf(-x);
    return 1.0 / (1.0 + z);
  } else {
    float z = expf(x);
    return z / (1.0 + z);
  }
}

__global__ void gLSTMOutputBackward(float* outCell, float* outXW, float* outSU, float* outB, const float* cell, const float* xW, const float* sU, const float* b, const float* adj, size_t rows, size_t cols) {
  int j = blockIdx.x + blockIdx.y * gridDim.x; // optimized grid stride loop
  while (j < rows) {
    float* rowOutCell = outCell + j * cols;
    float* rowOutXW = outXW + j * cols * 4;
    float* rowOutSU = outSU + j * cols * 4;

    const float* rowCell = cell + j * cols;
    const float* xWrow = xW + j * cols * 4;
    const float* sUrow = sU + j * cols * 4;

    const float* rowAdj = adj + j * cols;

    int i = threadIdx.x + blockDim.x * blockIdx.z; // optimized block stride loop
    while (i < cols) {
      int k = i + 3 * cols;
      float go = stableSigmoid(xWrow[k] + sUrow[k] + b[k]);

      float t = tanhf(rowCell[i]);

      float adj = rowAdj[i];

      // dc/dc_{t-1}
      if(outCell)
        rowOutCell[i] += go * (1 - t * t) * adj;

      // dc/d(b_o) = dc/d(xW_f) ...
      float dcdxo = t * go * (1 - go) * adj;
      if(outXW)
        rowOutXW[k] += dcdxo;
      if(outSU)
        rowOutSU[k] += dcdxo;
      if(outB)
        atomicAdd(outB + k, dcdxo);

      i += blockDim.x * gridDim.z;
    }

    j += gridDim.x * gridDim.y;
  }
}