#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gPasteRows(float* out, const float* in, size_t cols, const size_t* targetRowIdx, size_t rows) {
    // Use shared memory to reduce global memory accesses
    extern __shared__ float sharedMem[];

    int j = blockIdx.x + blockIdx.y * gridDim.x;
    if (j < rows) {
        size_t dstId = targetRowIdx[j];
        size_t srcId = j;

        float* rowOut = out + dstId * cols;
        const float* rowIn = in + srcId * cols;

        // Load a block of data into shared memory
        for (int i = threadIdx.x; i < cols; i += blockDim.x) {
            sharedMem[i] = rowIn[i];
        }
        __syncthreads();

        // Perform atomic addition using shared memory data
        for (int i = threadIdx.x; i < cols; i += blockDim.x) {
            atomicAdd(rowOut + i, sharedMem[i]);
        }
    }
}