#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableSigmoid(float x) {
    if (x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gHighwayForward(float* out, const float* in1, const float* in2, const float* t, size_t length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (index < length) { // Check if index is within bounds
        float sigma = stableSigmoid(t[index]);
        out[index] = in1[index] * sigma + in2[index] * (1.f - sigma);
    }
}
```
