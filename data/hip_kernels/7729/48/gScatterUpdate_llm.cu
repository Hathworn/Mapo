#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gScatterUpdate(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if idx is out of sparseSize bounds
    if (idx >= sparseSize) return;

    int index = sparseIndices[idx] + offset;
    
    // Check bounds and perform update in a single conditional
    if (index >= 0 && index < denseSize) {
        denseData[index] = sparseData[idx];
    }
}