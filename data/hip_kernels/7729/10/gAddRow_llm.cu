#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gAddRow(float* out, const float* in, int length) {
    // Use blockIdx.x and gridDim.x to evenly distribute the workload across blocks
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index does not exceed the length
    if(index < length) {
        out[index] += in[index];
    }
}