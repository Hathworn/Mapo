#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline float stableSigmoid(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gLSTMCellBackward(float* outCell, float* outXW, float* outSU, float* outB, const float* cell, const float* xW, const float* sU, const float* b, const float* mask, const float* adj, size_t rows, size_t cols) {
    int j = blockIdx.x;
    if(j < rows) {
        float m = !mask || mask[j];
        float* rowOutCell = outCell + j * cols;
        float* rowOutXW = outXW + j * cols * 4;
        float* rowOutSU = outSU + j * cols * 4;
        
        const float* rowCell = cell + j * cols;
        const float* xWrow = xW + j * cols * 4;
        const float* sUrow = sU + j * cols * 4;
        const float* rowAdj = adj + j * cols;
        
        // Use shared memory for improved memory access
        extern __shared__ float shared[];
        float* sharedXW = shared;
        float* sharedSU = shared + cols * 4;

        // Load xW and sU to shared memory
        for (int tid = threadIdx.x; tid < cols * 4; tid += blockDim.x) {
            sharedXW[tid] = xWrow[tid];
            sharedSU[tid] = sUrow[tid];
        }
        __syncthreads();

        for(int i = threadIdx.x; i < cols; i += blockDim.x) {
            float gf = stableSigmoid(sharedXW[i] + sharedSU[i] + b[i]);
            
            int k = i + cols;
            float gi = stableSigmoid(sharedXW[k] + sharedSU[k] + b[k]);
            
            int l = i + 2 * cols;
            float gc = tanhf(sharedXW[l] + sharedSU[l] + b[l]);
            
            float adj = rowAdj[i];
            
            // dc/dc_{t-1}
            if(outCell)
                rowOutCell[i] += (m * gf - m + 1) * adj;
                
            // dc/d(b_f) = dc/d(xW_f) ...
            float dcdxf = m * rowCell[i] * gf * (1 - gf) * adj;
            if(outXW)
                rowOutXW[i] += dcdxf;
            if(outSU)
                rowOutSU[i] += dcdxf;
            if(outB)
                atomicAdd(outB + i, dcdxf);
            
            // dc/d(b_i) ...
            float dcdb_i = m * gc * gi * (1 - gi) * adj;
            if(outXW)
                rowOutXW[k] += dcdb_i;
            if(outSU)
                rowOutSU[k] += dcdb_i;
            if(outB)
                atomicAdd(outB + k, dcdb_i);
            
            // dc/d(b_c) ...
            float dcdxc = m * gi * (1 - gc * gc) * adj;
            if(outXW)
                rowOutXW[l] += dcdxc;
            if(outSU)
                rowOutSU[l] += dcdxc;
            if(outB)
                atomicAdd(outB + l, dcdxc);
        }
    }
}