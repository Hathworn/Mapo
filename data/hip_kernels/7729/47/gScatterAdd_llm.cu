#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gScatterAdd(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure only required threads perform operations
    if (idx < sparseSize) {
        int index = sparseIndices[idx] + offset;

        // Check bounds within the if statement to avoid calculating repeatedly
        if (index >= 0 && index < denseSize) {
            atomicAdd(&denseData[index], sparseData[idx]);
        }
    }
}