#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gMaxPoolingForward(float* out, int outRows, int outCols, float* in, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Early exit for threads outside the output matrix
    if(tid >= outRows * outCols) return;

    int rowId = tid / outCols;  // Corrected columns division for rowId
    int colId = tid % outCols;  // Corrected columns modulo for colId

    // Calculate the starting pointer for input and local mask for current thread
    float* b = in + (rowId * inCols) + (colId * width);
    float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;

    // Adjust width for the last column
    if(colId == outCols - 1) {
        width = lastWidth;
    }

    // Initialize maximum value calculation
    float currentMax = b[0] * localMask[0];
    for(int i = 1; i < width; ++i) {
        float product = b[i] * localMask[i];
        if(product > currentMax) {
            currentMax = product;  // Update maximum if current is greater
        }
    }

    // Output the maximum value for current thread
    out[rowId * outCols + colId] = currentMax;  // Correct index calculation for storing result
}