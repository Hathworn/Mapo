#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gArgmax(float* out, const float* data, size_t rows, size_t cols) {
    size_t row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= rows) return;  // Ensure thread accesses within bounds

    size_t startInd = row * cols;
    float maxScore = -FLT_MAX;  // Use FLT_MAX for better readability
    size_t maxInd = 0;  // Initialize maxInd correctly

    // Unroll loop to improve performance
    for (size_t col = 0; col < cols; col += 4) {
        size_t ind = startInd + col;
        
        float score = (col < cols) ? data[ind] : -FLT_MAX;
        if (score > maxScore) {
            maxScore = score;
            maxInd = col;
        }
        
        if (col + 1 < cols) {
            score = data[ind + 1];
            if (score > maxScore) {
                maxScore = score;
                maxInd = col + 1;
            }
        }
        
        if (col + 2 < cols) {
            score = data[ind + 2];
            if (score > maxScore) {
                maxScore = score;
                maxInd = col + 2;
            }
        }
        
        if (col + 3 < cols) {
            score = data[ind + 3];
            if (score > maxScore) {
                maxScore = score;
                maxInd = col + 3;
            }
        }
    }
    out[row] = maxInd;
}