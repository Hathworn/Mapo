#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gCopyCols(float* out, const float* in, size_t rows, size_t colsIn, const size_t* sourceColIdx, size_t colsOut) {
    int j = blockIdx.x + blockIdx.y * gridDim.x;
    if (j < rows) {
        const float* rowIn = in + j * colsIn;
        float* rowOut = out + j * colsOut;

        for (int i = threadIdx.x; i < colsOut; i += blockDim.x) {
            // Copy column value from source to destination
            rowOut[i] = rowIn[sourceColIdx[i]];
        }
    }
}