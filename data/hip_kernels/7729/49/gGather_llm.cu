#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gGather(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    // Calculate the global index
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure the thread operates within bounds
    if (idx < sparseSize) {
        int index = sparseIndices[idx] + offset;
        // Check bounds for accessing dense data
        if (index >= 0 && index < denseSize) {
            sparseData[idx] = denseData[index];
        }
    }
}