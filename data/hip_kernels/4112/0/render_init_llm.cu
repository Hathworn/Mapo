#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure that threads are within the bounds
    if (i < max_x && j < max_y) {
        int pixel_index = j * max_x + i;
        
        // Initialize random state for each pixel
        hiprand_init(pixel_index, 0, 0, &rand_state[pixel_index]);
    }
}