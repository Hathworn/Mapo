#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory to optimize memory access
__global__ void matrixMulCUDA(int *A, int *B, int *C) {
    // Shared memory for sub-matrices
    __shared__ int shared_A[13][13];
    __shared__ int shared_B[13][13];

    // Thread index
    int row = threadIdx.x;
    int col = threadIdx.y;

    int multi = 0;

    // Load elements into shared memory
    shared_A[row][col] = A[(row * 13) + col];
    shared_B[row][col] = B[(row * 13) + col];
    __syncthreads();

    // Perform multiplication using shared memory
    for (int j = 0; j < 13; j++) {
        multi += shared_A[row][j] * shared_B[j][col];
    }

    // Write the computed value to the result matrix
    C[(row * 13) + col] = multi + shared_A[row][col] + shared_B[row][col];
}