#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv_horizontal_naive_gradInput(const int n, float *dx, const float *dy, const float *w, const int oH, const int oW, const int kL)
{
    // Use shared memory to store weights for faster access
    extern __shared__ float shared_w[];
    
    for (int i = threadIdx.x; i < kL; i += blockDim.x) {
        shared_w[i] = w[blockIdx.x * kL + i];
    }
    __syncthreads();
    
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        int iW = oW + kL - 1;
        int col = i % iW;
        int dy_offset = (i / iW) * oW + col;
        
        int k_begin = max(0, col - oW + 1);
        int k_end = min(kL, col + 1);

        float sum = 0.0f;
        // Loop unrolling for improved throughput
        for (int k = k_begin; k < k_end; k++) {
            sum += shared_w[k] * dy[dy_offset - k];
        }
        dx[i] = sum;
    }
}