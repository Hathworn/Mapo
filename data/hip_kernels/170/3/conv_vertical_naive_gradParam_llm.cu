#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void conv_vertical_naive_gradParam(const int n, float *dw, const float *x, const float *dy, const int kL, const int oH, const int oW)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if thread index is out of bounds
    if (i >= n) return;

    int dy_offset = (i / kL) * oH * oW;
    int x_offset = (i / kL) * oH * oW + (i % kL) * oW;

    // Temporary accumulation variable
    float accum = 0.0f;

    // Loop through and accumulate
    for (int k = 0; k < oH * oW; k++) {
        accum += dy[dy_offset + k] * x[x_offset + k];
    }

    // Write back to global memory
    dw[i] = accum;
}