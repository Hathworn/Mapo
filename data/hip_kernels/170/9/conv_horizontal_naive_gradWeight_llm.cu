#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conv_horizontal_naive_gradWeight(const int n, float *y, const float *x, const int kL, const int iC)
{
    // Calculate index directly from thread and block index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (; i < n; i += stride) {
        // Direct computation of y[i] using precalculated indices
        y[i] = x[(i / kL) * kL * iC + i];
    }
}