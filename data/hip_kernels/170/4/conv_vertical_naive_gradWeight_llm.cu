#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv_vertical_naive_gradWeight(const int n, float *y, const float *x, const int kL, const int iC)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop to cover the entire array, increasing memory access efficiency
    int stride = blockDim.x * gridDim.x;
    for (; i < n; i += stride) {
        y[i] = x[(i/kL) * kL * iC + i];
    }
}