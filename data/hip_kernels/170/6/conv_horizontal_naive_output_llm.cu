#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv_horizontal_naive_output(const int n, float *y, const float *x, const float *w, const int iH, const int iW, const int kL)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread index doesn't exceed limit
    if (i < n) {
        int oW = iW - kL + 1;
        int x_offset = (i / oW) * iW + i % oW;
        int w_offset = (i / (oW * iH)) * kL;
        float sum = 0.0f;

        // Iterate over convolution kernel length and accumulate results
        for (int k = 0; k < kL; k++) {
            sum += w[w_offset + k] * x[x_offset + k];
        }

        // Storing sum result into output array
        y[i] = sum;
    }
}