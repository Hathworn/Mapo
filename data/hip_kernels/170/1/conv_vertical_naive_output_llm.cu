#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv_vertical_naive_output(const int n, float *y, const float *x, const float *w, const int iH, const int iW, const int kL)
{
    // Use shared memory to cache input values and weights for faster access
    extern __shared__ float cache[]; 
    float* w_cache = cache; 
    float* x_cache = &cache[kL];

    int oH = iH - kL + 1;
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = global_idx; i < n; i += stride) {
        // Compute offsets
        int x_offset = (i / (oH * iW)) * iH * iW + i % (oH * iW);
        int w_offset = (i / (oH * iW)) * kL;

        // Load weight into shared memory
        if (threadIdx.x < kL) {
            w_cache[threadIdx.x] = w[w_offset + threadIdx.x];
        }
        __syncthreads();

        // Accumulate product into result
        float result = 0.0f;
        for (int k = 0; k < kL; k++) {
            // Load input into shared memory
            if (threadIdx.x < iW) {
                x_cache[threadIdx.x] = x[x_offset + k * iW + threadIdx.x];
            }
            __syncthreads();

            result += w_cache[k] * x_cache[i % iW];
            __syncthreads();
        }

        y[i] = result;
    }
}