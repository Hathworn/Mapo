#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv_vertical_naive_gradInput(const int n, float *dx, const float *dy, const float *w, const int oH, const int oW, const int kL) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop striding optimization
    for (; i < n; i += stride) {

        int iH = oH + kL - 1;
        int iC = i / (iH * oW);
        int row = (i % (iH * oW)) / oW;
        int dy_offset = iC * oH * oW + i % (iH * oW);
        int w_offset = iC * kL;

        int k_begin = max(0, row - oH + 1);
        int k_end = min(kL, row + 1);

        float sum = 0.0f;
        // Reducing global memory access
        for (int k = k_begin; k < k_end; k++) {
            sum += w[w_offset + k] * dy[dy_offset - k * oW];
        }
        dx[i] = sum;
    }
}