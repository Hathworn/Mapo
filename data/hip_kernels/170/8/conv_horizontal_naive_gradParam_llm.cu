#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv_horizontal_naive_gradParam(const int n, float *dw, const float *x, const float *dy, const int kL, const int oH, const int oW)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int iW = oW + kL - 1;

    for (int i = idx; i < n; i += stride) {
        int dy_offset = (i / kL) * oH * oW;
        int x_offset = (i / kL) * oH * oW + i % kL;
        float sum = 0.0f;  // Initialize local sum for reduction

        for (int j = 0; j < oH; j++) {
            for (int k = 0; k < oW; k++) {
                sum += dy[dy_offset + j * oW + k] * x[x_offset + j * iW + k];
            }
        }
        dw[i] = sum;  // Store accumulated sum into global memory
    }
}