#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f; // Use a register for accumulation

    // Optimized loop to accumulate partial results in a register
    while (eidx < numElements) {
        sum += a[eidx] * b[eidx];
        eidx += gridDim.x * DP_BLOCKSIZE;
    }
    
    // Store the partial result in shared memory
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Optimize reduction using warp shuffling
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        if (threadIdx.x < 16) mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        if (threadIdx.x < 8) mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        if (threadIdx.x < 4) mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        if (threadIdx.x < 2) mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        if (threadIdx.x < 1) mysh[threadIdx.x] += mysh[threadIdx.x + 1];

        // First thread writes the result to the target
        if (threadIdx.x == 0) {
            target[blockIdx.x] = mysh[0];
        }
    }
}