#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    // Optimize using cooperative thread indexing to reduce calculations
    uint globalThreadId = blockDim.x * blockIdx.x + threadIdx.x;

    // Initialize the random state with the optimized thread index
    hiprand_init(seed, globalThreadId, 0, &state[globalThreadId]);
}