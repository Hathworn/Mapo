#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0;

    // Unroll the loop for better performance
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Use parallel reduction instead of serial reduction
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }

    tmp = reduction_buffer[0];

    // Unroll the loop for better performance
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}