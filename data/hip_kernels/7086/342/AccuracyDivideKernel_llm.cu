#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel with individual block reduction
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    __shared__ float cache[256]; // Shared memory cache

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;
    
    // Use warp-wide reduction for better efficiency
    float temp_acc = (index < N) ? 1.0 : 0.0;

    // First phase of reduction
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        temp_acc += __shfl_down_sync(0xFFFFFFFF, temp_acc, i);
    }

    // Store reduction result in shared memory
    if (tid == 0) {
        cache[blockIdx.x] = temp_acc;
    }
    __syncthreads();

    // Final reduction across blocks
    if (blockIdx.x == 0 && tid < N) {
        float final_acc = (tid < N) ? cache[tid] : 0.0;
        for (int i = 1; i < gridDim.x; i++) {
            final_acc += cache[i];
        }
        if (tid == 0) {
            *accuracy = final_acc / N;
        }
    }
}