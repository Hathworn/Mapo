#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Improved block and thread indexing
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Efficient loop using calculated stride
    for (uint i = idx; i < tgtWidth * tgtHeight; i += stride) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        // Calculate source indices with modulo for wrapping
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Map target to tiled source
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}