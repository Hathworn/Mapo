#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N (2048*2048)
#define N_THREADS_PER_BLOCK 512

// Optimize memory access by checking index validity within device 
__global__ void addByCombine(int *a, int *b, int *c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global index
    if (index < N) { // Ensure index is within bounds
        c[index] = a[index] + b[index];
    }
}