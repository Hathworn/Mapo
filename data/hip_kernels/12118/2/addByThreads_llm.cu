#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addByThreads(int *a, int *b, int *c)
{
    // Calculate the global index for each thread
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure index does not exceed array bounds
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}