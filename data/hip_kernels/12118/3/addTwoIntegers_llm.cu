#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel optimized for simplicity, assuming one thread will handle the operation
__global__ void addTwoIntegers(int *a, int *b, int *c)
{
    // Directly add and store result if we are using a single-threaded approach
    *c = *a + *b;
}