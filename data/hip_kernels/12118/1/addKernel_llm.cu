#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *a, int *b, int *c)
{
    // Calculate global index for the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use global index to perform operation
    c[i] = a[i] + b[i];
}