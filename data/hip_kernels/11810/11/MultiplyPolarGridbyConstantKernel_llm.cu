#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MultiplyPolarGridbyConstantKernel(double *Dens, int nrad, int nsec, double ScalingFactor)
{
    // Calculate global thread index for 2D grid and 2D block configuration
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure i is strictly less than nrad for safe memory access
    if (i < nrad && j < nsec) // Check grid bounds
        Dens[i * nsec + j] *= ScalingFactor;
}