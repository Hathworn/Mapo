#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void MinusMeanKernel(double *Dens, double *Energy, double SigmaMed, double mean_dens_r, double mean_dens_r2, double mean_energy_r, double mean_energy_r2, double EnergyMed, int nsec, int nrad, double SigmaMed2, double EnergyMed2)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Precompute common expressions
    double dens_adjustment = SigmaMed - mean_dens_r;
    double energy_adjustment = EnergyMed - mean_energy_r;
    double dens_adjustment2 = SigmaMed2 - mean_dens_r2;
    double energy_adjustment2 = EnergyMed2 - mean_energy_r2;

    if (j < nsec) {
        // Use computed values instead of recalculating
        Dens[j] += dens_adjustment;
        Energy[j] += energy_adjustment;
    }

    int i = nrad - 1;
    if (j < nsec) {
        // Use computed values instead of recalculating
        Dens[i * nsec + j] += dens_adjustment2;
        Energy[i * nsec + j] += energy_adjustment2;
    }
}