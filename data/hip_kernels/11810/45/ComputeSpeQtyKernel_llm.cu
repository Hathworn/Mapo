#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeSpeQtyKernel(double *Label, double *Dens, double *ExtLabel, int nrad, int nsec)
{
    // Calculate global thread ID
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    
    // Boundary check
    if (i < nrad && j < nsec) {
        // Main computation
        Label[i * nsec + j] = ExtLabel[i * nsec + j] / Dens[i * nsec + j];
        // Compressive flow possibility
        // Label[i * nsec + j] = ExtLabel[i * nsec + j];
    }
}