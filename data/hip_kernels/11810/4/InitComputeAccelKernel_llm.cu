#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InitComputeAccelKernel(double *CellAbscissa, double *CellOrdinate, double *Rmed, int nsec, int nrad) 
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i < nrad && j < nsec) {
        // Precompute common terms for efficiency
        double angle = (2.0 * PI * (double)j) / (double)nsec;
        double Rmed_i = Rmed[i];

        // Store results directly with fewer operations
        CellAbscissa[i * nsec + j] = Rmed_i * cos(angle);
        CellOrdinate[i * nsec + j] = Rmed_i * sin(angle);
    }
}