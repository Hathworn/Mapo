#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeVelocitiesKernel (double *Vrad, double *Vtheta, double *Dens, double *Rmed, double *ThetaMomP, double *ThetaMomM, double *RadMomP, double *RadMomM, int nrad, int nsec, double OmegaFrame)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < nrad && j < nsec) {
        // Use condition to avoid division by small numbers or zero
        double dens_ij = Dens[i*nsec + j];
        double dens_ijm1 = Dens[i*nsec + ((j - 1 + nsec) % nsec)];
        
        if (i == 0) {
            Vrad[i*nsec + j] = 0.0;
        } else {
            double dens_im1j = Dens[(i - 1)*nsec + j];
            double denominator_rad = dens_ij + dens_im1j + 1e-20;
            Vrad[i*nsec + j] = (RadMomP[(i - 1)*nsec + j] + RadMomM[i*nsec + j]) / denominator_rad;
        }
        
        double denominator_theta = dens_ij + dens_ijm1 + 1e-15;
        Vtheta[i*nsec + j] = (ThetaMomP[i*nsec + ((j - 1 + nsec) % nsec)] + ThetaMomM[i*nsec + j]) / denominator_theta / Rmed[i] - Rmed[i] * OmegaFrame;
    }
}