#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double min2(double a, double b)
{
    return b < a ? b : a;
}

__device__ double max2(double a, double b)
{
    return b > a ? b : a;
}

__global__ void ConditionCFLKernel2D1 (double *Rsup, double *Rinf, double *Rmed, int nsec, int nrad, double *Vresidual, double *Vtheta, double *Vmoy, int FastTransport, double *SoundSpeed, double *Vrad, double *DT2D)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i > 0 && i < nrad && j < nsec) {

        double dxrad = Rsup[i] - Rinf[i];
        double dxtheta = Rmed[i] * 2.0 * PI / (double)nsec;

        // Use ternary operator for simplicity and potential performance gain
        Vresidual[i * nsec + j] = FastTransport ? (Vtheta[i * nsec + j] - Vmoy[i]) : Vtheta[i * nsec + j];

        double invdt1 = SoundSpeed[i * nsec + j] / min2(dxrad, dxtheta);
        double invdt2 = fabs(Vrad[i * nsec + j]) / dxrad;
        double invdt3 = fabs(Vresidual[i * nsec + j]) / dxtheta;

        // Simplifying velocity difference calculation
        double dvr = Vrad[(i + 1) * nsec + j] - Vrad[i * nsec + j];
        double dvt = Vtheta[i * nsec + (j + 1) % nsec] - Vtheta[i * nsec + j];

        // Ensure non-negative values with a ternary operator
        dvr = dvr >= 0.0 ? 1e-10 : -dvr;
        dvt = dvt >= 0.0 ? 1e-10 : -dvt;

        double invdt4 = max2(dvr / dxrad, dvt / dxtheta);
        invdt4 *= 4.0 * CVNR * CVNR;
        
        double dt = CFLSECURITY / sqrt(invdt1 * invdt1 + invdt2 * invdt2 + invdt3 * invdt3 + invdt4 * invdt4);
        DT2D[i * nsec + j] = dt; // array nrad*nsec size dt
    }
}