#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeAverageThetaVelocitiesKernel(double *Vtheta, double *VMed, int nsec, int nrad)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for intermediate summation to improve memory access pattern
    extern __shared__ double sharedMoy[];
    
    if (i < nrad) {
        double moy = 0.0;
        for (int j = threadIdx.y; j < nsec; j += blockDim.y) {
            moy += Vtheta[i * nsec + j];
        }
        
        // Reduce within block to compute total moy for the segment
        sharedMoy[threadIdx.x] = moy;
        __syncthreads();
        
        if (threadIdx.x == 0) {
            moy = 0.0;
            for (int k = 0; k < blockDim.x; ++k) {
                moy += sharedMoy[k];
            }
            VMed[i] = moy / (double)nsec;
        }
    }
}