#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VanLeerThetaKernel(double *Rsup, double *Rinf, double *Surf, double dt, int nrad, int nsec, int UniformTransport, int *NoSplitAdvection, double *QRStar, double *DensStar, double *Vazimutal_d, double *Qbase)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i < nrad && j < nsec)
    {
        // Only perform computations when conditions are met
        if ((UniformTransport == NO) || (NoSplitAdvection[i] == NO))
        {
            double dxrad = (Rsup[i] - Rinf[i]) * dt;
            double invsurf = 1.0 / Surf[i];
            double varq = dxrad * QRStar[i * nsec + j] * DensStar[i * nsec + j] * Vazimutal_d[i * nsec + j];
            
            // Reduce redundant calculations
            int next_j = (j + 1) % nsec;
            varq -= dxrad * QRStar[i * nsec + next_j] * DensStar[i * nsec + next_j] * Vazimutal_d[i * nsec + next_j];
            
            Qbase[i * nsec + j] += varq * invsurf;
        }
    }
}