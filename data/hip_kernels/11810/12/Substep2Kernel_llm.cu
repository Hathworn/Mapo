#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Substep2Kernel (double *Dens, double *VradInt, double *VthetaInt, double *TemperInt, int nrad, int nsec, double *invdiffRmed, double *invdiffRsup, double *DensInt, int Adiabatic, double *Rmed, double dt, double *VradNew, double *VthetaNew, double *Energy, double *EnergyInt)
{
    // Use a single thread ID calculation to improve readability and performance
    int index = threadIdx.x + blockDim.x * blockIdx.x + (threadIdx.y + blockDim.y * blockIdx.y) * nsec;

    if (index < nrad * nsec) {
        int i = index / nsec;
        int j = index % nsec;

        double dv;

        // Calculate radial velocity difference
        dv = VradInt[(i + 1) * nsec + j] - VradInt[i * nsec + j];

        // Update DensInt using ternary operator for clarity
        DensInt[i * nsec + j] = (dv < 0.0) ? CVNR * CVNR * Dens[i * nsec + j] * dv * dv : 0.0;

        // Calculate azimuthal velocity difference
        dv = VthetaInt[i * nsec + (j + 1) % nsec] - VthetaInt[i * nsec + j];

        // Update TemperInt using ternary operator for clarity
        TemperInt[i * nsec + j] = (dv < 0.0) ? CVNR * CVNR * Dens[i * nsec + j] * dv * dv : 0.0;
    }
}