#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void StarThetaKernel(double *Qbase, double *Rmed, int nrad, int nsec, double *dq, double dt) {
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i < nrad && j < nsec) {
        // Optimize dxtheta calculation to avoid redundant condition
        double dxtheta = (i < nrad) ? 2.0 * PI / (double)nsec * Rmed[i] : 0.0;
        double invdxtheta = 1.0 / dxtheta;

        // Precompute indices to reduce redundant calculations
        int index = i * nsec + j;
        int indexPrev = i * nsec + ((j - 1) + nsec) % nsec;
        int indexNext = i * nsec + (j + 1) % nsec;

        double dqm = Qbase[index] - Qbase[indexPrev];
        double dqp = Qbase[indexNext] - Qbase[index];

        // Simplified condition for dq calculation
        dq[index] = (dqp * dqm > 0.0) ? (dqp * dqm / (dqp + dqm) * invdxtheta) : 0.0;
    }
}