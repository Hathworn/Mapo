#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ConditionCFLKernel2D2 (double *newDT, double *DT2D, double *DT1D, double *Vmoy, double *invRmed, int *CFL, int nsec, int nrad, double DeltaT)
{
  // Calculate 'i' index for the current thread using block and thread indices
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  double newdt = 1e30; // Initialize newdt to a large value instead of repeating inside the loop

  // Only proceed if 'i' is within valid range
  if (i < nrad) {
    if (i > 0) { // Ensure 'i' is greater than 0 to proceed with the inner loop
      newDT[i] = newdt; // Initialize newDT with a large value
      for (int k = 0; k < nsec; k++) {
        // Compare each DT2D value and update newDT with the minimum
        newDT[i] = fmin(newDT[i], DT2D[i * nsec + k]);
      }
    }
    if (i < nrad - 1) {
      // Calculate dt only when 'i' is within valid range
      double dt = 2.0 * PI * CFLSECURITY / (double)nsec / fabs(Vmoy[i] * invRmed[i] - Vmoy[i + 1] * invRmed[i + 1]);
      DT1D[i] = dt; // Update DT1D with calculated dt values
    }
  }
}