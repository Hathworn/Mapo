#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ExtQtyKernel(double *ExtLabel, double *Dens, double *Label, int nsec, int nrad)
{
    // Calculate global thread index in flatten grid
    int j = blockIdx.x * blockDim.x + threadIdx.x; 
    int i = blockIdx.y * blockDim.y + threadIdx.y; 

    // Boundary check to avoid out-of-bounds memory access
    if (i < nrad && j < nsec) 
    {
        // Compute extended quantity
        int index = i * nsec + j; // Compute the flattened index
        ExtLabel[index] = Dens[index] * Label[index]; // Efficient memory access
    }
}