#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Make1DprofileKernel (double *gridfield, double *axifield, int nsec, int nrad)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to store intermediate results for better memory access efficiency
    extern __shared__ double sdata[];

    if (i < nrad) {
        double sum = 0.0;
        int j;

        for (j = threadIdx.x; j < nsec; j += blockDim.x) {
            sum += gridfield[i * nsec + j];
        }

        sdata[threadIdx.x] = sum;
        __syncthreads();

        // Perform a reduction sum within the block
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                sdata[threadIdx.x] += sdata[threadIdx.x + s];
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            axifield[i] = sdata[0] / (double)nsec;
        }
    }
}
```
