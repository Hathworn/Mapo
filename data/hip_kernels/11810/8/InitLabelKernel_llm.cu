#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InitLabelKernel (double *Label, double xp, double yp, double rhill, double *Rmed, int nrad, int nsec)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Ensure indices are within bounds
    if (i < nrad && j < nsec) {
        // Compute angle, x, y values
        double angle = ((double)j / nsec) * 2.0 * PI;
        double x = Rmed[i] * cos(angle);
        double y = Rmed[i] * sin(angle);

        // Compute distance and label condition
        double dx = x - xp;
        double dy = y - yp;
        double distance = hypot(dx, dy); // Use hypot for numerical stability

        // Set Label based on distance comparison
        Label[i * nsec + j] = (distance < rhill) ? 1.0 : 0.0;
    }
}