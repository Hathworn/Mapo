#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VanLeerRadialKernel(double *Rinf, double *Rsup, double *QRStar, double *DensStar, double *Vrad, double *LostByDisk, int nsec, int nrad, double dt, int OpenInner, double *Qbase, double *invSurf)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = blockIdx.y;  // Use block y-dimension for better memory coalescing

    if (i < nrad && j < nsec) {
        double dtheta = 2.0 * PI / (double)nsec;
        int index = i * nsec + j;
        int indexNext = (i + 1) * nsec + j;
        
        // Cache frequently used calculations
        double rinf_qrstar_dens_vrad = Rinf[i] * QRStar[index] * DensStar[index] * Vrad[index];
        double rsup_qrstar_dens_vrad = Rsup[i] * QRStar[indexNext] * DensStar[indexNext] * Vrad[indexNext];
        
        // Calculate varq using cached values
        double varq = dt * dtheta * (rinf_qrstar_dens_vrad - rsup_qrstar_dens_vrad);

        // Update Qbase with calculated varq
        Qbase[index] += varq * invSurf[i];

        // Handle OpenInner condition for LostByDisk
        if (i == 0 && OpenInner) {
            LostByDisk[j] = varq;
        }
    }
}