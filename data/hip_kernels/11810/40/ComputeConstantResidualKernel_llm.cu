#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeConstantResidualKernel (double *VMed, double *invRmed, int *Nshift, int *NoSplitAdvection, int nsec, int nrad, double dt, double *Vtheta, double *VthetaRes, double *Rmed, int FastTransport)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i < nrad && j < nsec) {
        // Use ternary operator for maxfrac
        double maxfrac = FastTransport ? 1.0 : 0.0;

        double invdt = 1.0 / dt;
        double dpinvns = 2.0 * PI / (double)nsec;
        double Ntilde = VMed[i] * invRmed[i] * dt * (double)nsec / (2.0 * PI);
        double Nround = floor(Ntilde + 0.5);

        // Direct assignment to Nshift
        Nshift[i] = (long)Nround;

        double Vtheta_val = (Ntilde - Nround) * Rmed[i] * invdt * dpinvns;
        Vtheta[i * nsec + j] = Vtheta_val;
        
        // Use direct condition without intermediate variable for branching
        if (maxfrac < 0.5) {
            NoSplitAdvection[i] = YES;
            VthetaRes[i * nsec + j] += Vtheta_val;
            Vtheta[i * nsec + j] = 0.0;
        } else {
            NoSplitAdvection[i] = NO;
        }
    }
}