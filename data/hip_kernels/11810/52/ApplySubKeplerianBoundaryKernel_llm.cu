#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ApplySubKeplerianBoundaryKernel(double *VthetaInt, double *Rmed, double OmegaFrame, int nsec, int nrad, double VKepIn, double VKepOut)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Check if thread index is within bounds for more concise range checking
    if (j < nsec) {
        // Process both the first and last radial indices in a single kernel call for better efficiency
        VthetaInt[j] = VKepIn - Rmed[0] * OmegaFrame;                              // i = 0, streamline
        VthetaInt[(nrad - 1) * nsec + j] = VKepOut - Rmed[nrad - 1] * OmegaFrame;  // i = nrad - 1, streamline
    }
}