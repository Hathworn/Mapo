#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReduceMeanKernel(double *Dens, double *Energy, int nsec, double *mean_dens, double *mean_energy, double *mean_dens2, double *mean_energy2, int nrad)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Unroll loop by handling both sections in parallel if valid
    if (j < nsec) {
        // Precompute indexes
        int index0 = j;
        int index1 = (nrad - 1) * nsec + j;

        // Coalesce memory access by applying both mean calculations simultaneously
        mean_dens[j] = Dens[index0];
        mean_energy[j] = Energy[index0];
        mean_dens2[j] = Dens[index1];
        mean_energy2[j] = Energy[index1];
    }
}