#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CircumPlanetaryMassKernel (double *Dens, double *Surf, double *CellAbscissa, double *CellOrdinate, double xpl, double ypl, int nrad, int nsec, double HillRadius, double *mdcp0)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i < nrad && j < nsec) {
        double x = CellAbscissa[i * nsec + j] - xpl;
        double y = CellOrdinate[i * nsec + j] - ypl;
        double dist = sqrt(x * x + y * y); // Optimize distance computation

        if (dist < HillRadius) {
            mdcp0[i * nsec + j] = Surf[i] * Dens[i * nsec + j];
        } else {
            mdcp0[i * nsec + j] = 0.0;
        }
    }
}