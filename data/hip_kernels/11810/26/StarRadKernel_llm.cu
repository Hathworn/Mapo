#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void StarRadKernel(double *Qbase2, double *Vrad, double *QStar, double dt, int nrad, int nsec, double *invdiffRmed, double *Rmed, double *dq) {
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Use shared memory for frequently accessed data to reduce global memory reads
    __shared__ double shared_invdiffRmed[1024];
    if (i < nrad) {
        shared_invdiffRmed[i] = invdiffRmed[i];
    }
    __syncthreads();

    if (i < nrad && j < nsec) {
        if (i == 0 || i == nrad - 1) {
            dq[i + j * nrad] = 0.0;
        } else {
            // Cache Qbase2 in registers to reduce global memory access within loop
            double Q_current = Qbase2[i * nsec + j];
            double Q_prev = Qbase2[(i - 1) * nsec + j];
            double Q_next = Qbase2[(i + 1) * nsec + j];
            
            double dqm = (Q_current - Q_prev) * shared_invdiffRmed[i];
            double dqp = (Q_next - Q_current) * shared_invdiffRmed[i + 1];

            if (dqp * dqm > 0.0) {
                dq[i + j * nrad] = 2.0 * dqp * dqm / (dqp + dqm);
            } else {
                dq[i + j * nrad] = 0.0;
            }
        }
    }
}