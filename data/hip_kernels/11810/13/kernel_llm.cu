#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(double *Dens, double *VradInt, double *VthetaInt, double *TemperInt, int nrad, int nsec, double *invdiffRmed, double *invdiffRsup, double *DensInt, int Adiabatic, double *Rmed, double dt, double *VradNew, double *VthetaNew, double *Energy, double *EnergyInt)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i < nrad && j < nsec) {
        double dxtheta = 2.0 * PI / nsec * Rmed[i];
        double invdxtheta = 1.0 / dxtheta;

        if (i > 0) {
            // Combined conditions and computation for VradNew
            double dens = Dens[i*nsec + j] + Dens[(i-1)*nsec + j];
            double densint = DensInt[i*nsec+j] - DensInt[(i-1)*nsec + j];
            VradNew[i*nsec+j] = VradInt[i*nsec+j] - dt * 2.0 / dens * densint * invdiffRmed[i];
        }

        // Optimized computation for VthetaNew
        double dens2 = Dens[i*nsec + j] + Dens[i*nsec + ((j-1)+nsec) % nsec];
        double tempint = TemperInt[i*nsec+j] - TemperInt[i*nsec + ((j-1)+nsec) % nsec];
        VthetaNew[i*nsec + j] = VthetaInt[i*nsec + j] - dt * 2.0 / dens2 * tempint * invdxtheta;

        // If Adiabatic, update EnergyInt
        if (Adiabatic) {
            EnergyInt[i * nsec + j] = Energy[i * nsec + j] - dt * DensInt[i * nsec + j] * 
            (VradInt[(i+1) * nsec + j] - VradInt[i * nsec + j]) * invdiffRsup[i] - 
            dt * TemperInt[i * nsec + j] * (VthetaInt[i * nsec + (j+1)%nsec] - VthetaInt[i * nsec + j]) * invdxtheta;
        }
    }
}