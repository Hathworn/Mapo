#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CrashKernel(double *array, int nrad, int nsec, int Crash)
{
    // Calculate flattened global thread index
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    // Combined flattened index for array access
    int index = idy * nsec + idx;

    // Boundary check for both dimensions
    if (idy < nrad && idx < nsec) {
        // Use ternary operator for concise conditional assignment
        array[index] = (array[index] < 0.0) ? 1.0 : 0.0;
    }
}