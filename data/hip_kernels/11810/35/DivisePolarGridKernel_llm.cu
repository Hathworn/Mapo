#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DivisePolarGridKernel (double *Qbase, double *DensInt, double *Work, int nrad, int nsec)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Compute global row index
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Compute global column index

    // Ensure threads accessing valid elements
    if (i < nsec && j < nrad) 
    {
        Work[i * nrad + j] = Qbase[i * nrad + j] / (DensInt[i * nrad + j] + 1e-20);
    }
}