#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ConditionCFLKernel2D3(double *newDT, double *DT2D, double *DT1D, double *Vmoy, double *invRmed, int *CFL, int nsec, int nrad, double DeltaT)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Restrict kernel execution to thread 0 to remove branch divergence
    if (j != 0) return;

    double newdt = newDT[1];
    // Use shared memory to reduce global memory access latency
    __shared__ double minVals[256];

    // Load values into shared memory for reduction
    for (int i = 2; i < nrad; i += blockDim.x) {
        if (threadIdx.x + i < nrad) {
            minVals[threadIdx.x] = newDT[threadIdx.x + i];
        } else {
            minVals[threadIdx.x] = newdt;
        }
        __syncthreads();

        // Parallel reduction in shared memory
        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            if (threadIdx.x < stride) {
                minVals[threadIdx.x] = min(minVals[threadIdx.x], minVals[threadIdx.x + stride]);
            }
        }
        __syncthreads();

        newdt = min(newdt, minVals[0]);
    }

    // Apply the same reduction method for DT1D
    for (int i = 0; i < nrad-1; i += blockDim.x) {
        if (threadIdx.x + i < nrad-1) {
            minVals[threadIdx.x] = DT1D[threadIdx.x + i];
        } else {
            minVals[threadIdx.x] = newdt;
        }
        __syncthreads();

        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
            __syncthreads();
            if (threadIdx.x < stride) {
                minVals[threadIdx.x] = min(minVals[threadIdx.x], minVals[threadIdx.x + stride]);
            }
        }
        __syncthreads();

        newdt = min(newdt, minVals[0]);
    }

    newdt = min(newdt, DeltaT);
    CFL[0] = (int)(ceil(DeltaT / newdt));
}