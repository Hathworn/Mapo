#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputePressureFieldKernel(double *SoundSpeed, double *Dens, double *Pressure, int Adiabatic, int nrad, int nsec, double ADIABATICINDEX, double *Energy) {
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i < nrad && j < nsec) {
        int idx = i * nsec + j; // Compute once, reuse index
        if (!Adiabatic) {
            double speed = SoundSpeed[idx]; // Avoid duplicate memory access
            Pressure[idx] = Dens[idx] * speed * speed;
        } else {
            Pressure[idx] = (ADIABATICINDEX - 1.0) * Energy[idx];
        }
    }
}