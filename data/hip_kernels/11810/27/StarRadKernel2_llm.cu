#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void StarRadKernel2 (double *Qbase2, double *Vrad, double *QStar, double dt, int nrad, int nsec, double *invdiffRmed, double *Rmed, double *dq)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Ensure indices are within bounds
    if (i < nrad && j < nsec) {
        double deltaR = Rmed[i] - (i > 0 ? Rmed[i-1] : Rmed[i+1]);
        double term = deltaR - Vrad[i*nsec + j] * dt;
        double dq_value = dq[i == 0 ? j*nrad : i-1+j*nrad];
        
        // Conditional assignments utilizing ternary operator for simplicity
        QStar[i*nsec + j] = Qbase2[(i > 0 ? i-1 : i)*nsec + j] - term * 0.5 * dq_value;
    }

    // Ensuring QStar has zero value for specific conditions
    if (i == 0 && j < nsec) {
        QStar[j] = QStar[j+nsec*nrad] = 0.0;
    }
}