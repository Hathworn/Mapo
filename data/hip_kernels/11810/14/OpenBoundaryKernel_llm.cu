#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void OpenBoundaryKernel(double *Vrad, double *Dens, double *Energy, int nsec, double SigmaMed)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = 1;

    if (j < nsec) {
        int idx = i * nsec + j;
        Dens[(i - 1) * nsec + j] = Dens[idx]; // copy first ring into ghost ring
        Energy[(i - 1) * nsec + j] = Energy[idx];
        // Simplified and optimized access to Vrad
        Vrad[idx] = (Vrad[(i + 1) * nsec + j] > 0.0 || Dens[idx] < SigmaMed) ? 0.0 : Vrad[(i + 1) * nsec + j];
    }
}