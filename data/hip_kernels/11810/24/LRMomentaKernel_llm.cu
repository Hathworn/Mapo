#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LRMomentaKernel (double *RadMomP, double *RadMomM, double *ThetaMomP, double *ThetaMomM, double *Dens, double *Vrad, double *Vtheta, int nrad, int nsec, double *Rmed, double OmegaFrame)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i < nrad && j < nsec) {
        int idx = i * nsec + j;  // Precompute linear index

        RadMomP[idx] = Dens[idx] * Vrad[(i+1) * nsec + j]; // (i+1)*nsec
        RadMomM[idx] = Dens[idx] * Vrad[idx];

        double ROmega = Rmed[i] * OmegaFrame; // Precompute ROmega
        double DensRmed = Dens[idx] * Rmed[i]; // Precompute DensRmed

        ThetaMomP[idx] = DensRmed * (Vtheta[i * nsec + (j+1) % nsec] + ROmega);
        ThetaMomM[idx] = DensRmed * (Vtheta[idx] + ROmega);
    }
}