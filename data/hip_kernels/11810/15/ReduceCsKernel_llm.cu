#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReduceCsKernel(double *SoundSpeed, double *cs0, double *cs1, double *csnrm1, double *csnrm2, int nsec, int nrad)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Check once for boundaries to avoid duplication
    if (j < nsec) {
        // Load i=0 and i=nrad-1 values
        cs0[j] = SoundSpeed[j];
        cs1[j] = SoundSpeed[nsec + j];
        csnrm1[j] = SoundSpeed[(nrad - 1) * nsec + j];
        csnrm2[j] = SoundSpeed[(nrad - 2) * nsec + j];
    }
}