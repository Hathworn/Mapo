#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeResidualsKernel(double *VthetaRes, double *VMed, int nsec, int nrad, double *Vtheta) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread indices are within range
    if (i < nrad && j < nsec) {
        int idx = i * nsec + j; // Pre-compute index for efficiency
        VthetaRes[idx] = Vtheta[idx] - VMed[i]; // Compute residual
    }
}