#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NonReflectingBoundaryKernel2(double *Dens, double *Energy, int i_angle, int nsec, double *Vrad, double *SoundSpeed, double SigmaMed, int nrad, double SigmaMed2, int i_angle2)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    if (j < nsec) { // Ensure j is within bounds
        double Vrad_med;

        // Calculate for i = 1
        int idx_1 = 1 * nsec + j;
        Vrad_med = -SoundSpeed[idx_1] * (Dens[idx_1] - SigmaMed) / SigmaMed;
        Vrad[idx_1] = 2.0 * Vrad_med - Vrad[(1 + 1) * nsec + j];

        // Calculate for i = nrad - 1
        int idx_nrad_1 = (nrad - 1) * nsec + j;
        int idx_nrad_2 = (nrad - 2) * nsec + j;
        Vrad_med = SoundSpeed[idx_nrad_1] * (Dens[idx_nrad_2] - SigmaMed2) / SigmaMed2;
        Vrad[idx_nrad_1] = 2.0 * Vrad_med - Vrad[idx_nrad_2];
    }
}
```
