#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CorrectVthetaKernel (double *Vtheta, double domega, double *Rmed, int nrad, int nsec)
{
    // Calculate the global thread indices for x and y dimensions
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread operates within the bounds of the array
    if (i < nrad && j < nsec)
    {
        // Perform the main operation on the relevant element
        Vtheta[i * nsec + j] -= domega * Rmed[i]; // Simplified the operation
    }
}