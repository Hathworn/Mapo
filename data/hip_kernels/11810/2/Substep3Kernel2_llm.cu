#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Substep3Kernel2(double *Dens, double *Qplus, double *viscosity_array, double *TAURR, double *TAURP, double *TAUPP, double *DivergenceVelocity, int nrad, int nsec, double *Rmed, int Cooling, double *EnergyNew, double dt, double *EnergyMed, double *SigmaMed, double *CoolingTimeMed, double *EnergyInt, double ADIABATICINDEX, double *QplusMed) {
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    double den, num;

    if (i == 0 && j < nsec) {
        // Calculate heating source term Qplus for i=0 using efficient check
        if (viscosity_array[nrad - 1] != 0.0) {
            double ratio = Qplus[(i + 1) * nsec + j] / Qplus[(i + 2) * nsec + j];
            double log_r_ratio = log(Rmed[i] / Rmed[i + 1]) / log(Rmed[i + 1] / Rmed[i + 2]);
            Qplus[i * nsec + j] = Qplus[(i + 1) * nsec + j] * exp(log(ratio) * log_r_ratio);
        } else {
            Qplus[i * nsec + j] = 0.0;
        }
    }

    if (i < nrad && j < nsec) {
        // Update energy with source terms considering Cooling status
        if (!Cooling) {
            den = 1.0 + (ADIABATICINDEX - 1.0) * dt * DivergenceVelocity[i * nsec + j];
            num = dt * Qplus[i * nsec + j] + EnergyInt[i * nsec + j];
            EnergyNew[i * nsec + j] = num / den;
        } else {
            num = EnergyMed[i] * dt * Dens[i * nsec + j] / SigmaMed[i] +
                  CoolingTimeMed[i] * EnergyInt[i * nsec + j] +
                  dt * CoolingTimeMed[i] * (Qplus[i * nsec + j] - QplusMed[i] * Dens[i * nsec + j] / SigmaMed[i]);

            den = dt + CoolingTimeMed[i] + (ADIABATICINDEX - 1.0) * dt * CoolingTimeMed[i] * DivergenceVelocity[i * nsec + j];
            EnergyNew[i * nsec + j] = num / den;
        }
    }
}