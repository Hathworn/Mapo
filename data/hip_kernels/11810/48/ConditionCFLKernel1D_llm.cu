#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ConditionCFLKernel1D (double *Rsup, double *Rinf, double *Rmed, int nrad, int nsec, double *Vtheta, double *Vmoy)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    // Check boundary condition for valid index
    if (i < nrad) {
        double sum = 0.0;  // Temporary variable to accumulate sum

        // Unroll loop for better performance
        for (int j = 0; j < nsec; j++) {
            sum += Vtheta[i * nsec + j];  // Accumulate values
        }
        
        Vmoy[i] = sum / (double)nsec;  // Calculate mean value
    }
}