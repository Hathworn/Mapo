#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void sieveOfEratosthenesCUDA(char *primes, uint64_t max)
{
    __shared__ int shared_flag;  // Use shared memory to store a flag

    if (threadIdx.x == 0 && threadIdx.y == 0) 
    {
        primes[0] = 1;
        primes[1] = 1;
        
        for (int j = 4; j < max; j += 2)
        {
            primes[j] = 1;
        }
    }
    else
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        const uint64_t maxRoot = sqrt((double)max);
        
        if (index < maxRoot && primes[index] == 0 && index > 1)
        {
            for (int j = index * index; j < max; j += index)
            {
                if (j % blockDim.x == 0) 
                {
                    if (threadIdx.x == 0)
                    {
                        shared_flag = 1;  // Set flag in shared memory
                    }
                    __syncthreads();  // Synchronize threads
                }
                
                primes[j] = 1;
            }
        }
    }
}