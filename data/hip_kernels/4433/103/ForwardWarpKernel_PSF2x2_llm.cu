#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Compute bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // Decompose into integer pixel and fractions
    float px, py, dx = modff(cx, &px), dy = modff(cy, &py);
    int tx = (int)px, ty = (int)py;
    float value = src[image_row_offset + j];

    // Pre-calculate weight factors
    float weight_dx_dy = dx * dy;
    float weight_1dx_dy = (1.0f - dx) * dy;
    float weight_1dx_1dy = (1.0f - dx) * (1.0f - dy);
    float weight_dx_1dy = dx * (1.0f - dy);

    // Bottom right corner
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        _atomicAdd(dst + ty * image_stride + tx, value * weight_dx_dy);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight_dx_dy);
    }

    // Bottom left corner
    tx -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        _atomicAdd(dst + ty * image_stride + tx, value * weight_1dx_dy);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight_1dx_dy);
    }

    // Upper left corner
    ty -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        _atomicAdd(dst + ty * image_stride + tx, value * weight_1dx_1dy);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight_1dx_1dy);
    }

    // Upper right corner
    tx += 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        _atomicAdd(dst + ty * image_stride + tx, value * weight_dx_1dy);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight_dx_1dy);
    }
}