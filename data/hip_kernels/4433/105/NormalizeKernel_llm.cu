#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Cache width and height for boundary checks to avoid multiple calculations.
    const int i = threadIdx.y + blockDim.y * blockIdx.y;
    const int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Use shared memory to reduce global memory accesses.
    __shared__ float shared_norm_factor[32][32]; // Assuming block size max 32x32, adjust as needed.

    if (i < h && j < w) {
        const int pos = i * s + j;

        // Load normalization factors into shared memory.
        shared_norm_factor[threadIdx.y][threadIdx.x] = normalization_factor[pos];
        __syncthreads();

        float scale = shared_norm_factor[threadIdx.y][threadIdx.x];

        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

        image[pos] *= invScale;
    }
}