#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Calculate linear thread index
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for better occupancy
    while (pos < len) {
        d_res[pos] = d_src[pos] * scale;
        pos += blockDim.x * gridDim.x;
    }
}