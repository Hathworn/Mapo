#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vector_add(float *c, const float *a, const float *b, const int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global index
    if (i < n) {
        c[i] = a[i] + b[i];  // Perform addition if within bounds
    }
}