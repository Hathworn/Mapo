#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cn_pnpoly_reference_kernel(int *bitmap, float2 *points, float2 *vertices, int n) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread corresponds to a valid point index
    if (i < n) {
        int c = 0;
        float2 p = points[i];  // Access point data
        int k = VERTICES - 1;
        
        // Loop to process each edge of the polygon
        for (int j = 0; j < VERTICES; k = j++) {
            float2 vj = vertices[j];  // Access vertex data
            float2 vk = vertices[k];
            
            // Optimize slope calculation
            float dy = vk.y - vj.y;
            float slope = dy != 0 ? (vk.x - vj.x) / dy : 0.0f;

            // Check ray intersection with polygon edge
            if (((vj.y > p.y) != (vk.y > p.y)) && (p.x < slope * (p.y - vj.y) + vj.x)) {
                c = !c;
            }
        }
        
        // Update bitmap with result
        bitmap[i] = c;  // Store result
    }
}