#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void computearray_size(int* block_cntr_array, int* finalsize, int* orig_number_of_char) {
    *finalsize = 0;
    for (int i = 0; i < *orig_number_of_char; i++) {
        *finalsize += block_cntr_array[i];
    }
}

__device__ int char_huffman_table_gpu[MAX_CHAR][MAX_CHAR-1];

bool* d_bool = 0;
bool* h_bool = 0;

__global__ void final_compression(int *block_cntr_array, bool *compressedfile_array, bool *finalcompressed_array, int number_of_char) {
    int index_blocks = blockIdx.x * blockDim.x + threadIdx.x;
    int final_index = 0;

    if(index_blocks < number_of_char) {
        // Optimize accumulation using prefix sum
        for (int i = 0; i < index_blocks; i++) {
            final_index += block_cntr_array[i];
        }

        // Optimize memory access for better coalescing
        int start_pos = (blockIdx.x * blockDim.x + threadIdx.x) * 255;
        for (int i = 0; i < block_cntr_array[index_blocks]; i++) {
            finalcompressed_array[final_index + i] = compressedfile_array[start_pos + i];
        }
    }
}

__global__ void compress_file_gpu(unsigned char *d_input, bool *compressedfile_array, int *char_huffman_table2, int *block_cntr_array, int* d_last_byte_padding, int *finalsize, int *orig_number_of_char, int number_of_char) {
    int block_counter = 0;
    unsigned char input_char;
    int index_file = (blockIdx.x * blockDim.x + threadIdx.x) * 255;
    int index_blocks = blockIdx.x * blockDim.x + threadIdx.x;

    if(index_blocks < number_of_char) {
        input_char = d_input[index_blocks];

        for(int i = 0; i < (MAX_CHAR - 1); i++) {
            int bit_value = char_huffman_table2[input_char * 255 + i];

            // Avoid repeated array access
            if(bit_value == 0) {
                compressedfile_array[index_file + i] = false;
                block_counter++;
            } else if(bit_value == 1) {
                compressedfile_array[index_file + i] = true;
                block_counter++;
            } else {
                break;
            }
        }

        block_cntr_array[index_blocks] = block_counter;
        computearray_size(block_cntr_array, finalsize, orig_number_of_char);
    }
}