#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Precompute source pixel's position in target space
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // Calculate integer and fractional parts
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    // Convert to integer target coordinates
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);

    // Early exit if the source pixel contributes to no valid target pixels
    if (tx < -1 || tx > w || ty < -1 || ty > h) return;

    float value = src[image_row_offset + j];
    float weight;

    // Optimize bounds check by handling all cases in a single trajectory iteration
    #pragma unroll 4
    for (int offset = 0; offset < 4; ++offset)
    {
        int x = tx + ((offset >> 1) & 1);
        int y = ty + (offset & 1);

        if (x >= 0 && x < w && y >= 0 && y < h)
        {
            weight = ((offset & 1) ? dx : (1.0f - dx)) * (((offset >> 1) & 1) ? dy : (1.0f - dy));
            _atomicAdd(dst + y * image_stride + x, value * weight);
            _atomicAdd(normalization_factor + y * image_stride + x, weight);
        }
    }
}