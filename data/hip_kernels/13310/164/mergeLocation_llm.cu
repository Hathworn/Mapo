#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate global thread index once
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure processing only for valid points
    if (ptidx < npoints)
    {
        // Read loc_ into local memory only once
        short2 loc = loc_[ptidx];

        // Utilize registers for scaled values to avoid repeated multiplication
        float scaledX = loc.x * scale;
        float scaledY = loc.y * scale;
        
        // Store the results directly
        x[ptidx] = scaledX;
        y[ptidx] = scaledY;
    }
}