#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpuSummationReduce(float *in, float *out, int n)
{
    extern __shared__ float sdata[];

    // Load shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? in[i] : 0;
    __syncthreads();

    // Optimized reduction in shared memory using contiguous threads
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s]; // Efficient reduction
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) out[blockIdx.x] = sdata[0];
}