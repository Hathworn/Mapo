#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FullToCOO(int numElem, float* H_vals, double* hamilValues, int dim)
{
    // Use shared memory to reduce global memory access for improved efficiency
    extern __shared__ float sharedH_vals[];

    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < numElem)
    {
        // Load data into shared memory for coalesced access
        sharedH_vals[threadIdx.x] = H_vals[i];
        __syncthreads();

        // Copy data from shared memory to hamilValues
        hamilValues[i] = sharedH_vals[threadIdx.x];
    }
}