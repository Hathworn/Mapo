#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_kernel(const float *x, const float *mean, const float *var, const float *weight, const float *bias, float *y, float *z, float eps, int N, int C, int S) {
    int plane = blockIdx.x;

    // Cache mean and variance for the current plane
    float _mean = mean[plane];
    float _var = var[plane];
    
    // Calculate inverse standard deviation and consider epsilon for numerical stability
    float invStd = rsqrtf(_var + eps);
    
    // Calculate gamma and beta for the current plane
    float gamma = (weight != nullptr) ? fabsf(weight[plane]) + eps : 1.f;
    float beta = (bias != nullptr) ? bias[plane] : 0.f;
    
    int globalIdx = (blockIdx.x * blockDim.x + threadIdx.x); // Global thread index
    int threadCount = blockDim.x * N; // Total number of threads
    
    // Use a single loop with stride across batches and spatial size
    for (int idx = globalIdx; idx < N * S; idx += threadCount) {
        int batch = idx / S;
        int n = idx % S;
        
        float _x = x[(batch * C + plane) * S + n]; // Load input
        float _y = (_x - _mean) * invStd;          // Normalize
        float _z = _y * gamma + beta;              // Scale and shift
        
        y[(batch * C + plane) * S + n] = _y;       // Write output y
        z[(batch * C + plane) * S + n] = _z;       // Write output z
    }
}