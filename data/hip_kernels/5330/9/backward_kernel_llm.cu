#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_kernel(const float *dz, const float *z, const float *var, const float *weight, const float *bias, const float *edz, const float *eydz, float *dx, float *dweight, float *dbias, float eps, int N, int C, int S) {
    int plane = blockIdx.x;
    float _edz = edz[plane];
    float _eydz = eydz[plane];

    float gamma = weight ? abs(weight[plane]) + eps : 1.f;
    float beta = bias ? bias[plane] : 0.f;

    if (dx) {
        float _var = var[plane];
        float invStd = (_var != 0.f || eps != 0.f) ? 1 / sqrt(_var + eps) : 0.f;
        float mul = gamma * invStd;

        int batchOffset = C * plane * S;
        for (int batch = 0; batch < N; ++batch) {
            int baseIdx = batch * batchOffset;
            for (int n = threadIdx.x; n < S; n += blockDim.x) {
                float _dz = dz[baseIdx + n];
                float _y = (z[baseIdx + n] - beta) / gamma;
                dx[baseIdx + n] = (_dz - _edz - _y * _eydz) * mul;
            }
        }
    }

    if (dweight || dbias) {
        float norm = N * S;

        if (dweight && threadIdx.x == 0) {
            float modifier = (weight[plane] > 0) ? _eydz * norm : (weight[plane] < 0) ? -_eydz * norm : 0;
            dweight[plane] += modifier;
        }

        if (dbias && threadIdx.x == 0) {
            dbias[plane] += _edz * norm;
        }
    }
}