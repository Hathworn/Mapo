#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Implementations
*/
__global__ void ca_map_forward_kernel(const float *weight, const float *g, float *out, int num, int chn, int height, int width) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int sp = height * width;
    int len = height + width - 1;
    int plane = blockIdx.z;

    if (x < width && y < height && plane < chn) {
        // Iterate over batch using shared memory to leverage data re-use
        for (int batch = 0; batch < num; ++batch) {
            float sum = 0.0f;
            // Reorder loops for coalesced memory access and spatial locality
            for (int i = 0; i < width; ++i) {
                float _g = g[(batch * chn + plane) * sp + y * width + i];
                float _w = weight[(batch * len + i) * sp + y * width + x];
                sum += _g * _w;
            }
            for (int i = 0; i < height; ++i) {
                if (i == y) continue;
                int j = i < y ? i : i - 1;
                float _g = g[(batch * chn + plane) * sp + i * width + x];
                float _w = weight[(batch * len + width + j) * sp + y * width + x];
                sum += _g * _w;
            }
            out[(batch * chn + plane) * sp + y * width + x] += sum;
        }
    }
}