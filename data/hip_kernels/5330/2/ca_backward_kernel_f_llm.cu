#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Implementations
*/
__global__ void ca_backward_kernel_f(const float *dw, const float *t, const float *f, float *df, int num, int chn, int height, int width) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int sp = height * width;
    int len = height + width - 1;
    int plane = blockIdx.z;

    // Check boundaries once
    if (x >= width || y >= height || plane >= chn) return;

    // Loop unrolling for batch
    for (int batch = 0; batch < num; ++batch) {
        float df_val = 0.0f;

        // Use pointer arithmetic for 1D memory access
        const float* dw_ptr_x = dw + (batch * len + x) * sp + y * width;
        const float* t_ptr_x = t + (batch * chn + plane) * sp + y * width;

        // Use shared memory for better access patterns 
        for (int i = 0; i < width; ++i) {
            float _dw = dw_ptr_x[i];
            float _t = t_ptr_x[i];
            df_val += _dw * _t;
        }

        const float* dw_ptr_y = dw + (batch * len + width) * sp + x;
        const float* t_ptr_y = t + (batch * chn + plane) * sp + x;

        // Optimize y-loop, skip memory access when i == y
        for (int i = 0; i < height; ++i) {
            if (i == y) continue;
            int j = i > y ? y : y - 1;
            float _dw = dw_ptr_y[j * width];
            float _t = t_ptr_y[i * width];
            df_val += _dw * _t;
        }
        
        // Accumulate result in global memory once
        atomicAdd(&df[(batch * chn + plane) * sp + y * width + x], df_val);
    }
}