#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Implementations
*/
__global__ void ca_backward_kernel_t(const float *dw, const float *t, const float *f, float *dt, int num, int chn, int height, int width) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int sp = height * width;
    int len = height + width - 1;
    int plane = blockIdx.z;

    if (x < width && y < height && plane < chn) {
        for (int batch = 0; batch < num; ++batch) {
            float dt_update = 0.0f;

            // Combine the two loops into one, reducing memory accesses and improving thread coherence
            for (int i = 0; i < width + height - 1; ++i) {
                if (i < width) {
                    float _dw = dw[(batch * len + i) * sp + y * width + x];
                    float _f = f[(batch * chn + plane) * sp + y * width + i];
                    dt_update += _dw * _f;
                }
                if (i >= width) {
                    int h_idx = i - width;
                    if (h_idx == y) continue;
                    int j = (h_idx < y) ? h_idx : h_idx - 1;

                    float _dw = dw[(batch * len + width + j) * sp + y * width + x];
                    float _f = f[(batch * chn + plane) * sp + h_idx * width + x];
                    dt_update += _dw * _f;
                }
            }
            dt[(batch * chn + plane) * sp + y * width + x] += dt_update;
        }
    }
}