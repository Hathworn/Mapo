#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void ca_map_backward_kernel_g(const float *dout, const float *weight, const float *g, float *dg, int num, int chn, int height, int width) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int sp = height * width;
    int len = height + width - 1;
    int plane = blockIdx.z;

    if (x < width && y < height && plane < chn) {
        for (int batch = 0; batch < num; ++batch) {
            // Unroll width loop with persistent threads
            for (int i = 0; i < width; ++i) {
                float _dout = dout[(batch * chn + plane) * sp + y * width + i];
                float _w = weight[(batch * len + x) * sp + y * width + i];
                atomicAdd(&dg[(batch * chn + plane) * sp + y * width + x], _dout * _w);
            }

            // Unroll height loop with merged calculations
            for (int i = 0; i < height; ++i) {
                if (i != y) {
                    int j = (i > y) ? y : y - 1;
                    float _dout = dout[(batch * chn + plane) * sp + i * width + x];
                    float _w = weight[(batch * len + width + j) * sp + i * width + x];
                    atomicAdd(&dg[(batch * chn + plane) * sp + y * width + x], _dout * _w);
                }
            }
        }
    }
}