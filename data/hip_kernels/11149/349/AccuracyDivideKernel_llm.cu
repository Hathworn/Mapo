#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
  // Using a single thread to perform the division for simplicity and efficiency
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    *accuracy /= N;
  }
}