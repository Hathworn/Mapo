#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // Use reduction for inner products with loop unrolling
    for (int i = idx; i < dim; i += blockDim.x * 4) {
        float sum = 0.0f;
        if (i < dim) sum += dY[i] * Y[i];
        if (i + blockDim.x < dim) sum += dY[i + blockDim.x] * Y[i + blockDim.x];
        if (i + 2 * blockDim.x < dim) sum += dY[i + 2 * blockDim.x] * Y[i + 2 * blockDim.x];
        if (i + 3 * blockDim.x < dim) sum += dY[i + 3 * blockDim.x] * Y[i + 3 * blockDim.x];
        tmp += sum;
    }

    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Perform parallel reduction
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (idx < s) {
            reduction_buffer[idx] += reduction_buffer[idx + s];
        }
        __syncthreads();
    }

    // Compute gradient
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}