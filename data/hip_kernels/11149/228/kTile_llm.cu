#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    // Calculate total elements once
    const uint numEls = tgtWidth * tgtHeight;

    for (uint i = idx; i < numEls; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        
        // Use modulo to wrap around source image
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        
        // Efficient direct indexing
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}