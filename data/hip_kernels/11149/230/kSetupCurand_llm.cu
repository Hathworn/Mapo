#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Use shared memory to reduce the number of unnecessary global memory accesses
    __shared__ hiprandState sharedState[NUM_RND_THREADS_PER_BLOCK];

    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    // Initialize hiprand state in shared memory
    hiprand_init(seed, tidx, 0, &sharedState[threadIdx.x]);

    // Copy the initialized state back to global memory
    state[tidx] = sharedState[threadIdx.x];
}