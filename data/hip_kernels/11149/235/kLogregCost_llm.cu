#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Calculate probability of correct prediction
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            // Use shared memory to reduce global memory accesses and improve performance.
            __shared__ int maxCountShared[LOGREG_ERR_THREADS_X];
            numMax = 0;

            for (int i = threadIdx.x; i < numOut; i += LOGREG_ERR_THREADS_X) {
                numMax += (probs[i * numCases + tx] == maxp);
            }

            // Reduce within block to calculate numMax
            maxCountShared[threadIdx.x] = numMax;
            __syncthreads();

            // Sum up results from shared memory
            if (threadIdx.x == 0) {
                numMax = 0;
                for (int i = 0; i < LOGREG_ERR_THREADS_X; i++) {
                    numMax += maxCountShared[i];
                }
                correctProbs[tx] = 1.0f / float(numMax);
            }
        }
    }
}