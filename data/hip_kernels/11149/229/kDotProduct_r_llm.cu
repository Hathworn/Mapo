#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0;  // Use a register for partial sum

    while (eidx < numElements) {
        sum += a[eidx] * b[eidx];  // Accumulate partial products in a register
        eidx += gridDim.x * DP_BLOCKSIZE;
    }

    shmem[threadIdx.x] = sum;  // Store the partial sum in shared memory
    __syncthreads();

    // Unroll the loop for reduction
    if (threadIdx.x < 128) {
        shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    }
    __syncthreads();
    if (threadIdx.x < 64) {
        shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    }
    __syncthreads();
    if (threadIdx.x < 32) {
        volatile float* mysh = &shmem[threadIdx.x];
        mysh[0] += mysh[32];
        mysh[0] += mysh[16];
        mysh[0] += mysh[8];
        mysh[0] += mysh[4];
        mysh[0] += mysh[2];
        mysh[0] += mysh[1];
    }

    // Only the first thread writes the result to the target array
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}