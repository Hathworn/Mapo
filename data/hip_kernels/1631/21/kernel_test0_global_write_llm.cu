#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int* orig_ptr = ptr;
    unsigned int pattern = 1;
    unsigned long mask = 4;

    *ptr = pattern;

    // Use unrolled loop to improve performance
    while (ptr < end_ptr)
    {
        ptr = (unsigned int*)(((unsigned long)orig_ptr) | mask);
        if (ptr >= end_ptr)
        {
            break;
        }
        *ptr = pattern;
        pattern <<= 1;  // Simplify pattern left shift
        mask <<= 1;     // Simplify mask left shift

        // Additional iteration to enhance throughput
        unsigned int* next_ptr = (unsigned int*)(((unsigned long)orig_ptr) | mask);
        if (next_ptr < end_ptr)
        {
            next_ptr = (unsigned int*)(((unsigned long)orig_ptr) | mask);
            *next_ptr = pattern;
            pattern <<= 1;  // Simplify pattern left shift
            mask <<= 1;     // Simplify mask left shift
        }
    }
    return;
}