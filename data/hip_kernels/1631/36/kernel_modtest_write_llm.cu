#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    // Cache block size and modulo size
    const unsigned int block_size = BLOCKSIZE / sizeof(unsigned int);
    const unsigned int* end_ptr_converted = (unsigned int*)end_ptr;

    // Calculate starting pointer for each block
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);

    // Prevent any out-of-bound memory access
    if (ptr >= end_ptr_converted) {
        return;
    }

    // Unroll loops for improved performance
    #pragma unroll
    for (unsigned int i = offset; i < block_size; i += MOD_SZ) {
        ptr[i] = p1;  // Write p1 at specific offset intervals
    }

    #pragma unroll
    for (unsigned int i = 0; i < block_size; i++) {
        if (i % MOD_SZ != offset) {
            ptr[i] = p2;  // Write p2 for other indices
        }
    }
}