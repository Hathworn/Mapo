#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Exit early if pointer is beyond end
    if (ptr >= (unsigned long*) end_ptr) {
        return;
    }

    // Use threadIdx.x to parallelize memory writes within block
    unsigned int idx = threadIdx.x;
    while (idx < BLOCKSIZE / sizeof(unsigned long)) {
        ptr[idx] = (unsigned long)&ptr[idx];
        idx += blockDim.x;
    }
}