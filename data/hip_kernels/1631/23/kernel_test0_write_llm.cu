#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    unsigned int* orig_ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* ptr = orig_ptr;
    
    // Check if pointer exceeds the end pointer at the beginning
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);
    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Write first pattern outside the loop for clarity
    *ptr = pattern;

    // Optimize loop condition and avoid unnecessary type casting
    while (true) {
        ptr = (unsigned int*) (((unsigned long)orig_ptr) | mask);
        
        if (ptr >= block_end) {
            break;
        }

        // Avoid unnecessary writes by checking pointer change
        if (ptr == orig_ptr) {
            mask <<= 1;
            continue;
        }

        *ptr = pattern;
        pattern <<= 1;
        mask <<= 1;
    }

    return;
}