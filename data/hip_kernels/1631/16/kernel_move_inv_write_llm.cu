#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    // Calculate the starting pointer for this block
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Exit early if the block is out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }
    
    // Calculate the iteration limit
    unsigned int limit = BLOCKSIZE / sizeof(unsigned int);
    
    // Use a warp-synchronous loop for better parallelism
    for (unsigned int i = threadIdx.x; i < limit; i += blockDim.x) {
        ptr[i] = pattern;
    }
    
    return;
}