#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    // Compute global index and allocate shared memory for pattern
    unsigned int global_idx = blockIdx.x * BLOCKSIZE / sizeof(unsigned int) + threadIdx.x;
    __shared__ unsigned int shared_pattern[BLOCKSIZE/sizeof(unsigned int)];

    // Convert char pointers to unsigned int pointers
    unsigned int* ptr = (unsigned int*)_ptr;

    // Exit early if out of bounds
    if (ptr + global_idx >= (unsigned int*)end_ptr) {
        return;
    }

    // Initialize variables
    unsigned int k = offset;
    unsigned int pat = pattern;

    // Write data to shared memory pattern
    shared_pattern[threadIdx.x] = pat;

    // Update pattern in shared memory
    k++;
    if (k >= 32) {
        k = 0;
        shared_pattern[threadIdx.x] = lb;
    } else {
        shared_pattern[threadIdx.x] = shared_pattern[threadIdx.x] << 1;
        shared_pattern[threadIdx.x] |= sval;
    }

    // Synchronize to ensure all writes to shared memory are done
    __syncthreads();

    // Write from shared memory to global memory
    ptr[global_idx] = shared_pattern[threadIdx.x];
}