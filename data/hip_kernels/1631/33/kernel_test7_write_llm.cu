#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* start_ptr = (unsigned int*) _start_ptr;

    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Use threadIdx to distribute work across threads and optimize memory access
    int stride = blockDim.x * gridDim.x;  // Calculate the stride for each thread
    for (int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += stride) {
        ptr[i] = start_ptr[i];
    }
}