#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* end_pos = (unsigned int*) end_ptr;

    // Early exit if pointer exceeds end
    if (ptr >= end_pos) {
        return;
    }

    // Use an optimized loop to apply pattern
    #pragma unroll 
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        ptr[i] = pattern;
    }
}