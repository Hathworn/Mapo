#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void increment_kernel(int *g_data, int inc_value) {
    // Compute global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform increment operation, potentially reducing access latency when memory is aligned
    int value = g_data[idx];
    g_data[idx] = value + inc_value;
}