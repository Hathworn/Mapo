#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_vbo_kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{
    // Calculate global thread ID
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within bounds
    if (x < width && y < height) {
        // Calculate uv coordinates
        float u = (x / (float)width) * 2.0f - 1.0f;
        float v = (y / (float)height) * 2.0f - 1.0f;

        // Calculate simple sine wave pattern
        float freq = 4.0f;
        float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

        // Write output vertex
        pos[y * width + x] = make_float4(u, w, v, 1.0f);
    }
}