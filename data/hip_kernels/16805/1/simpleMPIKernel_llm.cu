#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for better performance using shared memory
__global__ void simpleMPIKernel(float *input, float *output) {
    extern __shared__ float sharedInput[];
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int localTid = threadIdx.x;

    // Load input into shared memory
    sharedInput[localTid] = input[tid];
    __syncthreads();

    // Perform computation using shared memory
    output[tid] = sqrt(sharedInput[localTid]);
}