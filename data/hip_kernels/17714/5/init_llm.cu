#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init(uint32_t *v, uint32_t val, uint32_t n) {
    // Use local variable for thread index calculation
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        v[i] = val;
    }
    // Reduce conditional divergence by placing printf after initialization
    if (i == 0) {
        printf("init\n");
    }
}