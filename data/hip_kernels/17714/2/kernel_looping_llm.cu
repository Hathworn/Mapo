#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_looping(float *point, unsigned int num) {
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int stride = gridDim.x * blockDim.x; // Compute stride once for efficiency

    // Loop unrolling optimization
    for (int iloop = 0; iloop < NLOOPS; ++iloop) {
        size_t offset = idx;
        while (offset < num) {
            point[offset] += 1;
            offset += stride;
        }
    }
}