#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize: Use shared memory to reduce contention on global memory
__global__ void analyze(const float *input, float *sum, int numElements) {
    extern __shared__ float sharedSum[];  // Declare shared memory
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int local_i = threadIdx.x;

    // Initialize shared memory
    if (local_i < numElements) {
        sharedSum[local_i] = 0;
    }
    __syncthreads();

    // Reduce the number of atomic operations
    if (i < numElements) {
        atomicAdd(&sharedSum[local_i], input[i]);
    }
    __syncthreads();

    // Write back to global memory
    if (local_i < numElements) {
        atomicAdd(sum + local_i, sharedSum[local_i]);
    }
}