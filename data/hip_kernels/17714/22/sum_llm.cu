#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(const float *input, float *output, int numElements) {
    // Get the index of the element to be processed by this thread
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Initialize shared memory for block-wide reduction
    extern __shared__ float sharedBuf[];
    
    // Load input data into shared memory, ensuring index is within bounds
    sharedBuf[threadIdx.x] = (index < numElements) ? input[index] : 0.0f;
    __syncthreads();
    
    // Perform reduction using shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            sharedBuf[threadIdx.x] += sharedBuf[threadIdx.x + stride];
        }
        __syncthreads();
    }
    
    // Write the block result to the output array
    if (threadIdx.x == 0) {
        atomicAdd(output, sharedBuf[0]);
    }
}