#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    c[idx] = a[idx] + b[idx]; // Perform addition using global index
}