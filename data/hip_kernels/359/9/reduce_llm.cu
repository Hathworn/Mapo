#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void reduce(float *g_idata, float *g_odata, unsigned int n)
{
    extern __shared__ float sdata[];

    // Improved global memory access by using strided indexing
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    float mySum = 0.0f;

    // Better loop control with additional bounds check
    if (i < n) {
        mySum = g_idata[i];
        if (i + blockDim.x < n) {
            mySum += g_idata[i + blockDim.x];
        }
    }

    // Each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();

    // Optimize shared memory reduction using loop unrolling
    if (blockDim.x >= 1024 && tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads();
    if (blockDim.x >= 512 && tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads();
    if (blockDim.x >= 256 && tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
    if (blockDim.x >= 128 && tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();

    if (tid < 32) {
        // Use warp-synchronous programming for efficiency
        volatile float* smem = sdata;
        smem[tid] += smem[tid + 32];
        smem[tid] += smem[tid + 16];
        smem[tid] += smem[tid + 8];
        smem[tid] += smem[tid + 4];
        smem[tid] += smem[tid + 2];
        smem[tid] += smem[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}