#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void matsub(float* X, float* Y, unsigned int size) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Perform subtraction if thread index is within bounds
        X[idx] -= Y[idx];
    }
}