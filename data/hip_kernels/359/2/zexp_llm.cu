#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void zexp(float* Z, float* X, float* Y, unsigned int size)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Unrolling for performance improvement
    for (unsigned long int i = idx; i < size; i += stride * 4) {
        Z[i] = X[i] * __expf(Y[i]);
        if (i + stride < size) Z[i + stride] = X[i + stride] * __expf(Y[i + stride]);
        if (i + 2 * stride < size) Z[i + 2 * stride] = X[i + 2 * stride] * __expf(Y[i + 2 * stride]);
        if (i + 3 * stride < size) Z[i + 3 * stride] = X[i + 3 * stride] * __expf(Y[i + 3 * stride]);
    }
}