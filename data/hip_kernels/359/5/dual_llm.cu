#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void dual(float* err, float* Y, float* X, float* Z, unsigned int size) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    float local_err = 0.0f;

    // Loop with stride
    for (unsigned int i = idx; i < size; i += stride) {
        float temp = X[i] - Z[i]; // Compute difference
        Y[i] += temp; // Update Y array
        local_err += temp * temp; // Accumulate local error
    }

    // Atomic addition to ensure correct reduction
    atomicAdd(&err[0], local_err);
}