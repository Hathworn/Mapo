#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void colNorm_b(float* X, float* v, float* b, unsigned int size, unsigned int n)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    // Cache v and b in shared memory for faster access
    __shared__ float v_shared[256]; // Assuming n <= 256
    __shared__ float b_shared[256];

    if (threadIdx.x < n) {
        v_shared[threadIdx.x] = v[threadIdx.x];
        b_shared[threadIdx.x] = b[threadIdx.x];
    }
    __syncthreads();

    for (unsigned long int i = idx; i < size; i += stride) {
        unsigned int col = i % n;
        X[i] /= v_shared[col] * b_shared[col]; // Use cached values
    }
}