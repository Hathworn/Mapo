#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void vecInit(float* X, unsigned int size, float value)
{
    // Calculate the global thread ID
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop through indices with a stride of total threads
    for (unsigned int i = idx; i < size; i += blockDim.x * gridDim.x) {
        X[i] = value; // Set the value at each index
    }
}