#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void xexp(float* X, float* C, float* Y, float* Z, unsigned int size)
{
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (idx < size) {
        X[idx] = Z[idx] * __expf(C[idx] - Y[idx]);
    }
}