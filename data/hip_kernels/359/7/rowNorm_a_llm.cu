#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void rowNorm_a( float* X, float* v, float* a, unsigned int size, unsigned int n)
{
    // Calculate flat thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Use these values outside the loop since they are constant for each row
    unsigned int row;
    float v_val, a_val;

    if (idx < size) { // Check if thread should participate
        row = idx / n; // Pre-calculate row index
        v_val = v[row];
        a_val = a[row];

        for (unsigned long int i = idx; i < size; i += blockDim.x * gridDim.x) {
            // Efficiently use pre-calculated values
            X[i] /= v_val * a_val;
        }
    }
}