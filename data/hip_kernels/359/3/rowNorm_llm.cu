#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void rowNorm(float* X, float* v, unsigned int size, unsigned int n)
{
    // Calculate thread-specific index and stride
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Iterate with computed stride for workload balance
    for (unsigned int i = idx; i < size; i += stride) {
        unsigned int row = i / n;  // Calculate row index based on linear index
        X[i] /= v[row];  // Normalize current element by corresponding row value
    }
}