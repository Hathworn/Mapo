#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

// Optimized global kernel function
__global__ void colNorm(float* X, float* v, unsigned int size, unsigned int n)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use block-level cache to minimize global memory access for `v`
    __shared__ float v_shared[256]; // Assume a fixed max block size

    for (unsigned long int i = idx; i < size; i += stride) {
        // Load v[col] into shared memory only once
        if (threadIdx.x < n) {
            v_shared[threadIdx.x] = v[threadIdx.x];
        }
        __syncthreads();

        unsigned int col = i % n;
        X[i] /= v_shared[col];

        __syncthreads();
    }
}