#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dense_kernel(int num_input, int num_output, double* gpu_in, double* weights, double* biases, double* gpu_out, int num_classes)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= num_output) return;

    double sum = biases[tid]; // Initialize with bias directly
    // Loop unrolling for optimization
    int count = 0;
    for (; count <= num_input - 4; count += 4) {
        sum += gpu_in[count] * weights[tid * num_input + count];
        sum += gpu_in[count + 1] * weights[tid * num_input + count + 1];
        sum += gpu_in[count + 2] * weights[tid * num_input + count + 2];
        sum += gpu_in[count + 3] * weights[tid * num_input + count + 3];
    }
    for (; count < num_input; count++) {
        sum += gpu_in[count] * weights[tid * num_input + count];
    }
    
    // Use ternary operator for compact ReLU activation
    gpu_out[tid] = (num_output != num_classes && sum < 0.0) ? 0.0 : sum;
}