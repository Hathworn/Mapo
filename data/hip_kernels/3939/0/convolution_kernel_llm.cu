#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolution_kernel(int h, int w, int d, double* gpu_in, int k_h, int k_w, int k_d, double* kernel_weights, double* kernel_biases, int num_kernels, int op_h, int op_w, int op_d, double* gpu_out)
{
    // Identifying threads by their IDs
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int deep = blockDim.z * blockIdx.z + threadIdx.z;

    // Return if thread out of bounds
    if (row >= op_h || col >= op_w || deep >= op_d) return;

    double out = 0.0;

    // Avoid repeated calculations; directly compute the offsets
    int base_input_idx = h * w * deep;
    int base_kernel_idx = deep * k_h * k_w * k_d;

    #pragma unroll  // Use loop unrolling for potential performance increase
    for (int depth_pointer = 0; depth_pointer < k_d; ++depth_pointer) {
        int depth_offset = depth_pointer * h * w;
        for (int row_pointer = 0; row_pointer < k_h; ++row_pointer) {
            int row_offset = row_pointer * w;
            for (int column_pointer = 0; column_pointer < k_w; ++column_pointer) {
                out += gpu_in[(row * w + col) + column_pointer + row_offset + depth_offset] * kernel_weights[column_pointer + row_pointer * k_w + depth_pointer * k_h*k_w + base_kernel_idx];
            }
        }
    }

    // Bias addition and ReLU activation
    double result = out + kernel_biases[deep];
    gpu_out[row * op_w + col + deep * op_h * op_w] = (result < 0.0) ? 0.0 : result;
}