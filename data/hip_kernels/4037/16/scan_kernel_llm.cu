#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan_kernel(unsigned int* d_bins, int size) {
    int mid = threadIdx.x + blockDim.x * blockIdx.x;
    if (mid >= size) return;

    for (int s = 1; s <= size; s *= 2) {
        int spot = mid - s;
        
        // Use shared memory to reduce global memory access latency
        __shared__ unsigned int shared_bins[1024]; // Adjust size as needed based on block size
        shared_bins[threadIdx.x] = d_bins[mid];
        __syncthreads(); // Ensure all threads have written to shared memory
        
        unsigned int val = 0;
        if (spot >= 0) val = shared_bins[threadIdx.x - s];
        __syncthreads(); // Sync before writing back to global memory
        
        if (spot >= 0) d_bins[mid] += val;
        __syncthreads();
    }
}