#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
    // Initialize shared memory with input values
    s[threadIdx.x] = p;
    __syncthreads();

    // Perform warp-level reduction using binary tree method
    for (unsigned int stride = warpSize / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            s[threadIdx.x] += s[threadIdx.x + stride];
        }
        __syncthreads();
    }
    
    // Store and return result from shared memory
    return s[0];
}

__global__ void reduce(unsigned int * d_out_shared, const unsigned int * d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];

    // Call shared_reduce to compute sum within warp
    unsigned int sr = shared_reduce(p, s);

    // Only the first thread writes the result
    if (t == 0) {
        *d_out_shared = sr;
    }
}