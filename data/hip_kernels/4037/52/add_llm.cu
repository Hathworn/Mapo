#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int n, float *x, float *y) {
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use global index to perform parallel addition
    if (index < n) {
        y[index] = x[index] + y[index];
    }
}