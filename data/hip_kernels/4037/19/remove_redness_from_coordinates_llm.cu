#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void remove_redness_from_coordinates(
    const unsigned int* d_coordinates, 
    unsigned char* d_r, 
    unsigned char* d_b, 
    unsigned char* d_g, 
    unsigned char* d_r_output, 
    int num_coordinates, 
    int num_pixels_y, 
    int num_pixels_x, 
    int template_half_height, 
    int template_half_width) 
{
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_index_1d < num_coordinates) 
    {
        // Load coordinates and calculate positions
        unsigned int image_index_1d = d_coordinates[num_pixels_x * num_pixels_y - global_index_1d - 1];
        ushort2 image_index_2d = make_ushort2(image_index_1d % num_pixels_x, image_index_1d / num_pixels_x);

        // Unroll loops for better performance
        for (int y = image_index_2d.y - template_half_height; y <= image_index_2d.y + template_half_height; y++) 
        {
            int clamped_y = min(num_pixels_y - 1, max(0, y));
            
            for (int x = image_index_2d.x - template_half_width; x <= image_index_2d.x + template_half_width; x++) 
            {
                int clamped_x = min(num_pixels_x - 1, max(0, x));
                int image_offset_index_1d_clamped = num_pixels_x * clamped_y + clamped_x;
                
                // Read values and compute average
                unsigned int gb_average = (d_g[image_offset_index_1d_clamped] + d_b[image_offset_index_1d_clamped]) >> 1;
                d_r_output[image_offset_index_1d_clamped] = static_cast<unsigned char>(gb_average);
            }
        }
    }
}