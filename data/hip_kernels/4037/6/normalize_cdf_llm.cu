#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalize_cdf( unsigned int* d_input_cdf, float* d_output_cdf, int n )
{
    const float normalization_constant = 1.f / d_input_cdf[n - 1];

    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_index_1d < n)
    {
        // Use shared memory for frequently accessed data
        __shared__ float shared_constant;
        if (threadIdx.x == 0) {
            shared_constant = normalization_constant;
        }
        __syncthreads();

        unsigned int input_value = d_input_cdf[global_index_1d];
        float output_value = input_value * shared_constant;
        d_output_cdf[global_index_1d] = output_value;
    }
}