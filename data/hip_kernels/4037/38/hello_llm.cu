#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use placeholders for both block and thread indices
__global__ void hello()
{
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello world! I'm thread %d in block %d\n", globalThreadId, blockIdx.x);
}