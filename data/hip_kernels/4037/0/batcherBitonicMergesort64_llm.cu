#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// http://en.wikipedia.org/wiki/Bitonic_sort

__global__ void batcherBitonicMergesort64(float *d_out, const float *d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    for (int stage = 0; stage <= 5; stage++)
    {
        // Set mask to identify elements within this stage
        int mask = (1 << (stage + 1)) - 1;

        for (int substage = stage; substage >= 0; substage--)
        {
            // Calculate partner index
            int partner = tid ^ (1 << substage);

            // Ensure partner is within bounds and works in synchronized threads
            if (partner < 64)
            {
                // Compare and swap
                if ((tid & mask) == 0)
                {
                    if (sdata[tid] > sdata[partner])
                    {
                        // Swap if out of order
                        float temp = sdata[tid];
                        sdata[tid] = sdata[partner];
                        sdata[partner] = temp;
                    }
                }
                else
                {
                    if (sdata[tid] < sdata[partner])
                    {
                        // Swap if out of order
                        float temp = sdata[tid];
                        sdata[tid] = sdata[partner];
                        sdata[partner] = temp;
                    }
                }
            }
            __syncthreads(); // Ensure all swaps complete before next substage
        }
    }
    d_out[tid] = sdata[tid]; // Output the sorted data
}