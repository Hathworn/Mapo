#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Load thread index into a register for faster access
    int idx = threadIdx.x; 

    // Use printf within the kernel to display thread index
    printf("Hello world! I'm thread %d\n", idx);
}