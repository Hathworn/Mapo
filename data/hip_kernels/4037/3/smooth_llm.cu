#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth(float * v_new, const float * v) {
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x; // Correct thread index calculation
    int numThreads = blockDim.x * gridDim.x;
    
    // Using shared memory for performance improvement
    extern __shared__ float s[];
    s[threadIdx.x] = v[myIdx];

    if (threadIdx.x > 0 && threadIdx.x < blockDim.x - 1) {
        s[threadIdx.x - 1] = v[myIdx - 1];
        s[threadIdx.x + 1] = v[myIdx + 1];
    }

    __syncthreads(); // Synchronize threads to ensure shared memory is fully populated

    float myLeftElt = (myIdx > 0) ? s[threadIdx.x - 1] : v[myIdx]; // Avoid out-of-bounds
    float myRightElt = (myIdx < numThreads - 1) ? s[threadIdx.x + 1] : v[myIdx]; // Avoid out-of-bounds
    
    v_new[myIdx] = 0.25f * myLeftElt + 0.5f * s[threadIdx.x] + 0.25f * myRightElt;
}