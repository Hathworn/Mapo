#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int n, float *x, float *y) {
    // Calculate unique global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop through data using global indexing
    for (int i = index; i < n; i += stride) {
        y[i] = x[i] + y[i];
    }
}