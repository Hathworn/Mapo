#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int myItem = d_in[myId];
    if (myId < BIN_COUNT) // Ensure within bounds
    {
        int myBin = myItem % BIN_COUNT;
        atomicAdd(&(d_bins[myBin]), 1);
    }
}