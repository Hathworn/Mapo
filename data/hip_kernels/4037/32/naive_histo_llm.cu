#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void naive_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Store thread and block indices to shared memory to reduce global memory accesses
    __shared__ int shared_bins[1024]; 
    int myId = threadIdx.x + blockDim.x * blockIdx.x;

    // Initialize shared bins to zero in parallel
    if(threadIdx.x < BIN_COUNT) {
        shared_bins[threadIdx.x] = 0;
    }
    __syncthreads();

    // Find bin index using modulus operation
    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;
    
    // Atomic addition to avoid race condition
    atomicAdd(&(shared_bins[myBin]), 1);
    __syncthreads();

    // Write back to global memory
    if(threadIdx.x < BIN_COUNT) {
        atomicAdd(&(d_bins[threadIdx.x]), shared_bins[threadIdx.x]);
    }
}