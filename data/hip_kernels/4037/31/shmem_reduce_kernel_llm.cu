#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shmem_reduce_kernel(float *d_out, const float *d_in) {
    extern __shared__ float sdata[];

    // Calculate thread and block IDs
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // Load data into shared memory
    sdata[tid] = d_in[myId];
    __syncthreads();

    // Perform reduction using shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        // Make sure the threads accessing data are active
        if (tid < s) {
            // Use atomic operation to prevent race conditions
            atomicAdd(&sdata[tid], sdata[tid + s]);
        }
        __syncthreads(); 
    }

    // Write result from shared memory to global memory
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}