#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelGradf(const float *d_x, float *d_grad)
{
    // Load inputs into registers to reduce global memory access
    const float x0 = d_x[0];
    const float x1 = d_x[1];
    
    // Precompute reused expression to reduce arithmetic operations
    const float x0_squared = x0 * x0;
    const float temp = x1 - x0_squared;

    // df/dx0 = -2 (1-x0) - 400 (x1-x0^2) x0
    // df/dx1 = 200 (x1 - x0^2)
    d_grad[0] = -2.0f * (1.0f - x0) - 400.0f * x0 * temp;
    d_grad[1] = 200.0f * temp;
}