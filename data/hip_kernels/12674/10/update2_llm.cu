#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function
__global__ void update2(float *alphaMinusBeta_out, const float *rho, const float *yDotZ, const float *alpha)
{
    // Compute beta using shared variable to reduce global memory access
    float beta = (*rho) * (*yDotZ);
    
    // Compute result directly using loaded values
    *alphaMinusBeta_out = *alpha - beta;
}