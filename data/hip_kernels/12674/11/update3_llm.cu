#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update3(float *rho_out, float *H0_out, const float *yDotS, const float *yDotY)
{
    // Load data into registers to reduce global memory accesses
    float local_yDotS = *yDotS;
    float local_yDotY = *yDotY;

    // Compute and store *rho_out
    *rho_out = 1.0f / local_yDotS; 

    // Compute conditionally and store *H0_out
    if (local_yDotY > 1e-5)
    {
        *H0_out = local_yDotS / local_yDotY;
    }
}