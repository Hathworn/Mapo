#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelF(const float *d_x, float *d_y)
{
    // Use registers for a and b to optimize memory access
    const float x0 = d_x[0];
    const float x1 = d_x[1];
    float a = 1.0f - x0;
    float b = x1 - x0 * x0;

    // Store final value directly to d_y
    *d_y = a * a + 100.0f * b * b;
}