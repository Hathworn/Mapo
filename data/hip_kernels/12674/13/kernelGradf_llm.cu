#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelGradf(const float *d_x, float *d_grad, float *A, float *b, const size_t len)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= len)
        return;

    float sum = 0.0f; // Use register for accumulation to optimize memory access

    for (size_t j = 0; j < len; ++j)
    {
        sum += A[index * len + j] * d_x[j];
    }

    sum *= 2.0f; // Compute once in a register; reduces memory write operations
    d_grad[index] = sum + b[index];
}