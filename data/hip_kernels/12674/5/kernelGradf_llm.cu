#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelGradf(const float *d_x, float *d_grad, float *A, float *b, const size_t len)
{
    // Calculate global thread index
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    // Return if index exceeds bounds
    if (index >= len)
        return;

    float sum = 0.0f;

    // Use shared memory to optimize access to A
    extern __shared__ float shared_x[];
    shared_x[threadIdx.x] = d_x[threadIdx.x];
    __syncthreads();

    for (size_t j = 0; j < len; ++j)
    {
        sum += A[index * len + j] * shared_x[j];
    }

    // Directly store the result in global memory
    d_grad[index] = 2.0f * sum + b[index];
}