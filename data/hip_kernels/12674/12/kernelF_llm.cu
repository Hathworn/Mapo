#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelF(const float *d_xAx, const float *d_bx, const float *d_c, float *d_y)
{
    // Using threadIdx.x to allow parallel computation
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    d_y[idx] = d_xAx[idx] + d_bx[idx] + d_c[idx];
}