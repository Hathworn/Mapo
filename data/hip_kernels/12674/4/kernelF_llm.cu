#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelF(const float *d_xAx, const float *d_bx, const float *d_c, float *d_y)
{
    // Use threadIdx to parallelize summation if necessary
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx == 0) // Ensure single write to d_y since input is scalar
    {
        *d_y = *d_xAx + *d_bx + *d_c;
    }
}