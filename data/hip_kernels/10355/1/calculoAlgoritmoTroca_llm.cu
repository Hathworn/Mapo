#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculoAlgoritmoTroca(float *dev_matrizSuperior, int linhaPerm, int colunaPerm, int totalColunas, int totalLinhas)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Early exit for invalid or permissible row indices
    if (i >= totalLinhas || i == linhaPerm)
        return;

    float fatorAnulador = dev_matrizSuperior[i * totalColunas + colunaPerm] * (-1);

    // Calculate only valid elements within the row
    for (int coluna = 0; coluna < totalColunas; coluna++) {
        // Skip invalid memory access
        if (i * totalColunas + coluna >= totalLinhas * totalColunas)
            return;

        if (coluna == colunaPerm) {
            dev_matrizSuperior[i * totalColunas + coluna] = 0;
        } else {
            dev_matrizSuperior[i * totalColunas + coluna] = fatorAnulador
                * dev_matrizSuperior[linhaPerm * totalColunas + coluna]
                + dev_matrizSuperior[i * totalColunas + coluna];
        }
    }
}