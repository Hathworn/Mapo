#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void test_kernel()
{
    // Using cooperative grid synchronization
    __shared__ bool isLastBlockDone;
    if (threadIdx.x == 0) isLastBlockDone = false;
    __syncthreads();

    // Only first thread in each block performs operation to avoid congestion
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("test_kernel() - First Thread in Block\n");
    }
    
    __syncthreads();
    if (threadIdx.x == 0) {
        isLastBlockDone = true;
    }
    __syncthreads();

    // Sync all blocks
    if (isLastBlockDone) {
        if (threadIdx.x == 0 && blockIdx.x == 0) {
            printf("test_kernel() - All Blocks Synchronized\n");
        }
    }
}