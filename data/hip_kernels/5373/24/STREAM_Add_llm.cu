#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Add(float *a, float *b, float *c, size_t len)
{
    // Calculate global index
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid stride loop for processing
    for (size_t i = idx; i < len; i += blockDim.x * gridDim.x) {
        c[i] = a[i] + b[i];
    }
}