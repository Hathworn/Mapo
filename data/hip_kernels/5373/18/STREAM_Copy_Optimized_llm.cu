#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Copy_Optimized(float *a, float *b, size_t len)
{
    // Calculate global thread index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use striding to allow multiple iterations per thread
    size_t stride = blockDim.x * gridDim.x;
    
    for (size_t i = idx; i < len; i += stride)
    {
        b[i] = a[i];
    }
}