#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Scale_Optimized(float *a, float *b, float scale, size_t len)
{
    // Calculate global index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Efficient boundary check and perform scaling
    if (idx < len) 
    {
        b[idx] = scale * a[idx];
    }
}