#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Triad_Optimized_double(double *a, double *b, double *c, double scalar, size_t len)
{
    // Compute unique global thread index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Use stride loop for better occupancy and coalesced memory accesses
    for (size_t i = idx; i < len; i += blockDim.x * gridDim.x) {
        c[i] = a[i] + scalar * b[i];
    }
}