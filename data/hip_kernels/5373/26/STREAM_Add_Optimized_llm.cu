#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Add_Optimized(float *a, float *b, float *c, size_t len)
{
    // Calculate unique thread index for grid and block configuration
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Only perform computation if idx is within array bounds
    if (idx < len)
    {
        c[idx] = a[idx] + b[idx];
    }
}