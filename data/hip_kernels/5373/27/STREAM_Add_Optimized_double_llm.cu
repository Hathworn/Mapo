#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Add_Optimized_double(double *a, double *b, double *c, size_t len)
{
    // Calculate the global thread index
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use strided loop to ensure all elements are processed even with insufficient threads
    for (size_t i = idx; i < len; i += blockDim.x * gridDim.x) 
    {
        c[i] = a[i] + b[i];
    }
}