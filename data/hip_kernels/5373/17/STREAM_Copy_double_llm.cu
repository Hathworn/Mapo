#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Copy_double(double *a, double *b, size_t len)
{
    // Use const for limiting register usage and optimizing memory access
    const size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use stride loop for better memory coalescing
    for (size_t i = idx; i < len; i += blockDim.x * gridDim.x) {
        b[i] = a[i];
    }
}