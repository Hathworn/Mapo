#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Triad_double(double *a, double *b, double *c, double scalar, size_t len)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x; // Define stride outside the loop
    for (; idx < len; idx += stride) { // Use a for loop for clarity
        c[idx] = a[idx] + scalar * b[idx];
    }
}