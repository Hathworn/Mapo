#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Scale_double(double *a, double *b, double scale, size_t len)
{
    // Calculate global thread index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use stride loop for memory access
    size_t stride = blockDim.x * gridDim.x;
    
    for (; idx < len; idx += stride) {
        b[idx] = scale * a[idx];
    }
}