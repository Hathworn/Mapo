#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Scale(float *a, float *b, float scale, size_t len)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Optimize by using a strided loop and unrolling
    for (; idx < len; idx += blockDim.x * gridDim.x)
    {
        #pragma unroll
        b[idx] = scale * a[idx];
    }
}