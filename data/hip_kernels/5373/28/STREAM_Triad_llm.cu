#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Triad(float *a, float *b, float *c, float scalar, size_t len)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;
    for (; idx < len; idx += stride) {
        c[idx] = a[idx] + scalar * b[idx];
    }
}