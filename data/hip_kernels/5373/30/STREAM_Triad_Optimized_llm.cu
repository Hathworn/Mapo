#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Triad_Optimized(float *a, float *b, float *c, float scalar, size_t len)
{
    // Use stride loop to cover the entire data size
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = idx; i < len; i += stride) {
        c[i] = a[i] + scalar * b[i];
    }
}