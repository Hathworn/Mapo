#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Copy_Optimized_double(double *a, double *b, size_t len)
{
    // Calculate the unique thread index for each thread
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Use stride access pattern for better memory coalescing
    for (size_t i = idx; i < len; i += blockDim.x * gridDim.x) {
        b[i] = a[i];
    }
}