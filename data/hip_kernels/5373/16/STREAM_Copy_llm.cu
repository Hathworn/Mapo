#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Copy(float *a, float *b, size_t len)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Using strided loop for coalesced memory access
    for (; idx < len; idx += blockDim.x * gridDim.x) {
        b[idx] = a[idx];
    }
}