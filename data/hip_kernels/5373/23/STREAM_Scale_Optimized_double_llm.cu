#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Scale_Optimized_double(double *a, double *b, double scale, size_t len)
{
    // Use a more efficient approach by employing shared memory and unrolling
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;

    for (size_t i = idx; i < len; i += stride)
    {
        b[i] = scale * a[i];
    }
}