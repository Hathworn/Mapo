#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Add_double(double *a, double *b, double *c, size_t len)
{
    // Calculate global thread ID
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x; // Precompute stride

    // Loop unrolling for efficient memory access
    for (; idx + stride < len; idx += stride * 2) {
        c[idx] = a[idx] + b[idx];
        c[idx + stride] = a[idx + stride] + b[idx + stride];
    }

    // Handle remaining elements
    if (idx < len) {
        c[idx] = a[idx] + b[idx];
    }
}