#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_array_double(double *a, double value, size_t len)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;  // Pre-calculate stride for efficiency

    // Use a for loop for better readability and no need to calculate idx in each iteration
    for (; idx < len; idx += stride) {
        a[idx] = value;
    }
}