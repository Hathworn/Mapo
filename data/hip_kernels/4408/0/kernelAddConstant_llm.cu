#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// a simple kernel that increments each array element by b
__global__ void kernelAddConstant(int *g_a, const int b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure the index is within bounds for safe access
    if (idx < gridDim.x * blockDim.x) {
        g_a[idx] += b;
    }
}