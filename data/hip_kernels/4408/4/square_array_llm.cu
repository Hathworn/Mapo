#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    // Use shared memory for faster access
    __shared__ float shared_data[BLOCK_SIZE];
  
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
    {
        // Load data into shared memory
        shared_data[threadIdx.x] = a[idx];
        __syncthreads();
      
        // Perform the squaring operation
        shared_data[threadIdx.x] *= shared_data[threadIdx.x];
      
        __syncthreads();
      
        // Store the result back to global memory
        a[idx] = shared_data[threadIdx.x];
    }
}