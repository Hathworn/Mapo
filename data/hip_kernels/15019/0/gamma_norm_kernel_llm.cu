#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void gamma_norm_kernel(float* img, int image_height, int image_width, int image_step)
{
    // Calculate pixel positions using shared memory for better performance
    __shared__ int pixel_x_shared;
    __shared__ int pixel_y_shared;

    // Load pixel positions into shared memory
    if (threadIdx.x == 0) {
        pixel_x_shared = blockIdx.x * blockDim.y + threadIdx.y;
        pixel_y_shared = blockIdx.y * blockDim.z + threadIdx.z;
    }
    __syncthreads();

    // If current position is outside the image, stop here
    if(pixel_x_shared >= image_width || pixel_y_shared >= image_height)
    {
        return;
    }

    int channel = threadIdx.x;

    // Each row has image_step pixels and each pixel has three channels
    int in_pixel_idx = pixel_y_shared * image_step + pixel_x_shared * 3 + channel;

    // Finally perform the normalization
    img[in_pixel_idx] = sqrt(img[in_pixel_idx] / 256.0f);
}