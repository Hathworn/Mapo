#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogram_kernel(float* magnitude, float* phase, float* histograms, int input_width, int input_height, int cell_grid_width, int cell_grid_height, int magnitude_step, int phase_step, int histograms_step, int cell_width, int cell_height, int num_bins)
{
    // Shared memory for storing intermediate results
    __shared__ int s_lbin_pos[64];
    __shared__ float s_lbin[64];
    __shared__ int s_rbin_pos[64];
    __shared__ float s_rbin[64];
    __shared__ float s_hist[9 * 8];
    
    // Calculate pixel positions
    int pixel_x = blockIdx.x * blockDim.x + threadIdx.x;
    if(pixel_x >= input_width) return;
    int pixel_y = blockIdx.y * blockDim.y + threadIdx.y;
    if(pixel_y >= input_height) return;
    
    int mag_pixel_idx = pixel_y * magnitude_step + pixel_x;
    int phase_pixel_idx = pixel_y * phase_step + pixel_x;
    
    // Phase normalization
    float bin_size = 1.0f / (float)num_bins;
    int left_bin = (int)floor((phase[phase_pixel_idx] / bin_size) - 0.5f);
    left_bin = (left_bin + num_bins) % num_bins;
    int right_bin = (left_bin + 1);
    float delta = (phase[phase_pixel_idx] / bin_size) - right_bin;
    if(delta < -0.5) delta += num_bins;
    right_bin = right_bin % num_bins;

    // Store bin positions and amounts
    s_lbin_pos[threadIdx.x] = left_bin;
    s_lbin[threadIdx.x] = (0.5f - delta) * magnitude[mag_pixel_idx];
    s_rbin_pos[threadIdx.x] = right_bin;
    s_rbin[threadIdx.x] = (0.5f + delta) * magnitude[mag_pixel_idx];

    // Synchronize threads to ensure shared memory writes are complete
    __syncthreads();

    // Initialize shared histograms
    s_hist[threadIdx.x] = 0.0f;
    if(threadIdx.x < 8) s_hist[threadIdx.x + 64] = 0.0f;

    int cell_y = pixel_y / cell_height;

    // Calculate partial histograms in separate threads
    if(threadIdx.x < 8)
    {
        int s_hist_idx = 9 * threadIdx.x;
        for(int i = 1; i < 8; ++i)
        {
            s_hist[s_hist_idx + s_lbin_pos[8 * threadIdx.x + i]] += s_lbin[8 * threadIdx.x + i];
            s_hist[s_hist_idx + s_rbin_pos[8 * threadIdx.x + i]] += s_rbin[8 * threadIdx.x + i];
        }
    }

    // Synchronize threads before atomic operations
    __syncthreads();

    // Atomic addition to global histograms
    int out_idx = cell_y * histograms_step + threadIdx.x;
    atomicAdd(&(histograms[out_idx]), s_hist[threadIdx.x]);

    if(threadIdx.x < 8)
    {
        atomicAdd(&(histograms[out_idx + 64]), s_hist[threadIdx.x + 64]);
    }
}