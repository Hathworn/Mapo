#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void block_normalization_kernel(float* histograms, float* descriptor, int histograms_step, int block_grid_width, int block_grid_height, int block_width, int block_height, int num_bins, int cell_grid_width, int block_stride_x, int block_stride_y)
{
    // Shared memory for storing the histograms and L1 norms
    __shared__ float s_blocks[9 * 4 * 8];
    __shared__ float L1_norm[8];

    // Calculate block indices
    int block_x = blockIdx.x * 8 + threadIdx.z;
    int block_y = blockIdx.y;

    // Check for out-of-bound blocks
    if (block_x >= block_grid_width || block_y >= block_grid_height) {
        return;
    }

    // Compute indices for cell and histogram
    int cell_x = block_x * block_stride_x + threadIdx.y % 2;
    int cell_y = block_y * block_stride_y + threadIdx.y / 2;
    int hist_idx = histograms_step * cell_y + num_bins * cell_x + threadIdx.x;
    
    // Compute index for s_blocks
    int s_blocks_idx = 9 * threadIdx.y + threadIdx.x;
    // Load histogram data into shared memory
    s_blocks[s_blocks_idx] = histograms[hist_idx];

    __syncthreads();

    // Compute the L1 norm for the blocks in parallel
    int thread_id = 36 * threadIdx.z + 9 * threadIdx.y + threadIdx.x;
    if (thread_id < 8) {
        float sum = 0.0f;
        for (int i = 0; i < 36; ++i) {  // 36 is 9 bins * 4 cells per block
            sum += s_blocks[36 * thread_id + i];
        }
        L1_norm[thread_id] = sum;
    }

    __syncthreads();

    // Calculate descriptor by normalizing the histogram
    descriptor[histograms_step * (block_y * block_grid_width * 8 + block_x) + s_blocks_idx] =
        s_blocks[s_blocks_idx] / L1_norm[threadIdx.z];
}