#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread is within range
    if(i < n) {
        float p = pred[i];
        float t = truth[i];
        error[i] = (t != 0.0f) ? -logf(p) : 0.0f;  // Use logf for better performance
        delta[i] = t - p;
    }
}