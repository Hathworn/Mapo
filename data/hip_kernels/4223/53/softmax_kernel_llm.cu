#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    float sum = 0;
    float largest = -INFINITY;

    // Compute largest element for normalization
    for(int i = 0; i < n; ++i){
        float val = input[i * stride];
        largest = max(val, largest);
    }

    // Compute exponentials and sum
    for(int i = 0; i < n; ++i){
        float e = expf((input[i * stride] - largest) / temp);
        sum += e;
        output[i * stride] = e;
    }

    // Normalize the output
    float inv_sum = 1.0f / sum; // Precompute reciprocal of sum
    for(int i = 0; i < n; ++i){
        output[i * stride] *= inv_sum;
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D index calculation
    int stride_grid = gridDim.x * blockDim.x; // Total stride for the grid

    // Process multiple items per thread to increase occupancy
    for (int idx = id; idx < batch * groups; idx += stride_grid) {
        int b = idx / groups;
        int g = idx % groups;
        softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
    }
}