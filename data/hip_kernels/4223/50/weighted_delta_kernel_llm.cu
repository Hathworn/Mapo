#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate global thread index efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int gridSize = blockDim.x * gridDim.x;  // Calculate the grid size
    
    while (i < n) {
        // Perform calculations based on conditions
        if(da) da[i] += dc[i] * s[i];
        if(db) db[i] += dc[i] * (1 - s[i]);
        ds[i] += dc[i] * (a[i] - b[i]);

        i += gridSize;  // Move to the next element processed by this thread
    }
}