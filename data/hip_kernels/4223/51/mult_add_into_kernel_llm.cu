#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid stride loop to process all elements
    for (int index = i; index < n; index += blockDim.x * gridDim.x) {
        c[index] += a[index] * b[index];
    }
}