#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index for each thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by performing coalesced reads/writes
    int stride = blockDim.x * gridDim.x;

    // Loop with strided access to allow processing of larger arrays with fewer blocks
    for (; i < N; i += stride) {
        X[i * INCX] += ALPHA;
    }
}