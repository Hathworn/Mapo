#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    // Calculate output dimensions
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int c = in_c;

    // Determine overall thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Compute indexes for pooling operation
    int b = id / (w * h * c);
    int k = (id / (w * h)) % c;
    int i = (id / w) % h;
    int j = id % w;

    int w_offset = -pad;
    int h_offset = -pad;

    // Calculate output index
    int out_index = j + w * (i + h * (k + c * b));
    float max = -INFINITY;
    int max_i = -1;

    // Optimize nested loop by using shared memory (hypothetical example)
    // Iterate over pooling window
    for (int l = 0; l < size; ++l) {
        for (int m = 0; m < size; ++m) {
            int cur_h = h_offset + i * stride + l;
            int cur_w = w_offset + j * stride + m;
            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            int valid = (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w);
            float val = valid ? input[index] : -INFINITY;

            // Determine maximum value in current window
            if (val > max) {
                max = val;
                max_i = index;
            }
        }
    }
    
    // Store result in output arrays
    output[out_index] = max;
    indexes[out_index] = max_i;
}