#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0.0f;

    int filter = blockIdx.x;

    // Loop unrolling and accessing global memory efficiently
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += x[index];
        }
    }

    __syncthreads();

    // Use parallel reduction instead of sequential accumulation
    for (int offset = threads / 2; offset > 0; offset >>= 1) {
        if (id < offset) {
            local[id] += local[id + offset];
        }
        __syncthreads();
    }

    // Only one thread writes the result
    if (id == 0) {
        mean[filter] = local[0] / (spatial * batch);
    }
}