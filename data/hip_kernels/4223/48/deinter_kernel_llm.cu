#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = (NX + NY) * B;

    // Use a stride loop to process all necessary elements
    for (; i < total_elements; i += blockDim.x * gridDim.x) {
        int b = i / (NX + NY);
        int j = i % (NX + NY);

        if (j < NX) {
            if (X) X[b * NX + j] += OUT[i];  // Add OUT to X element
        } else {
            if (Y) Y[b * NY + j - NX] += OUT[i];  // Add OUT to Y element
        }
    }
}