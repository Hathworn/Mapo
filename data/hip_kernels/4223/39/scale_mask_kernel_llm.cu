#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int totalThreads = gridDim.x * blockDim.x; // Calculate total number of threads

    for(int idx = i; idx < n; idx += totalThreads) // Loop through elements with grid stride
    {
        if(mask[idx] == mask_num) // Check mask
        {
            x[idx] *= scale; // Apply scaling
        }
    }
}