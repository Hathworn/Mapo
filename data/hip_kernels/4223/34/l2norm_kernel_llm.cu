#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    // Optimize by using shared memory
    extern __shared__ float shared_sum[];

    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;

    int b = index / spatial;
    int i = index % spatial;
    int f;
    float sum = 0.0f;

    for(f = 0; f < filters; ++f) {
        int idx = b*filters*spatial + f*spatial + i;
        sum += x[idx] * x[idx];
    }

    // Use block-wise reduction for computing sum
    shared_sum[threadIdx.x] = sum;
    __syncthreads();

    sum = 0.0f;
    for(int j = 0; j < blockDim.x; ++j) {
        sum += shared_sum[j];
    }
    sum = sqrtf(sum);
    if(sum == 0) sum = 1;

    for(f = 0; f < filters; ++f) {
        int idx = b*filters*spatial + f*spatial + i;
        x[idx] /= sum;
        dx[idx] = (1 - x[idx]) / sum;
    }
}