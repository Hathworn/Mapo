#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate the linear index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a multiple of blockDim.x * gridDim.x for larger grids
    // Allow more threads or smaller grid size in case 'n' is small
    int total_threads = blockDim.x * gridDim.x;

    // Use a loop for processing multiple elements per thread
    for (; i < n; i += total_threads) {
        c[i] = s[i] * a[i] + (1 - s[i]) * (b ? b[i] : 0);
    }
}