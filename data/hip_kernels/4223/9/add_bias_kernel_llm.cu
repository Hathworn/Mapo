#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    // Calculate global index with threadIdx, blockIdx, blockDim, and gridDim
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Iterate through multiple outputs per thread
    int stride = blockDim.x * gridDim.x;
    
    for (; index < n * size * batch; index += stride) {
        int i = index % size;
        int j = (index / size) % n;
        int k = index / (n * size);

        // Update output by adding bias
        output[(k * n + j) * size + i] += biases[j];
    }
}