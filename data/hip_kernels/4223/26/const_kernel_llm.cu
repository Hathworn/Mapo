#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplify grid index calculation
    int stride = gridDim.x * blockDim.x; // Compute stride for grid-wide iteration
    for (int i = idx; i < N; i += stride) // Use loop to handle large arrays beyond single block size
    {
        X[i * INCX] = ALPHA; // Assign ALPHA to each element at strided positions
    }
}