#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = blockDim.x * gridDim.x;
    for(; i < n; i += gridStride) {
        float diff = truth[i] - pred[i];
        error[i] = fabsf(diff);  // Use vectorized operation for absolute value
        delta[i] = (diff > 0) ? 1.0f : -1.0f;  // Use float to ensure type consistency
    }
}