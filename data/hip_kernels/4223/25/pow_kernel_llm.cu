#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Using a 1D grid and 1D blocks to potentially improve warp execution efficiency
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Flatten the condition to improve branch prediction
    if (i < N) {
        Y[i * INCY] = pow(X[i * INCX], ALPHA);
    }
}