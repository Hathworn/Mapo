#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= N) return;
    
    // Calculate indices for output and input
    int out_index = i;
    int b = i / (w * h * c * stride * stride);
    int remainder = i % (w * h * c * stride * stride);
    int out_c = remainder / (w * h * stride * stride);
    remainder = remainder % (w * h * stride * stride);
    int out_h = remainder / (w * stride);
    int out_w = remainder % (w * stride);

    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_c = out_c;
    int in_index = b * w * h * c + in_c * w * h + in_h * w + in_w;

    // Perform operation based on direction
    if (forward) {
        out[out_index] += scale * x[in_index]; // Forward pass
    } else {
        atomicAdd(x + in_index, scale * out[out_index]); // Backward pass
    }
}