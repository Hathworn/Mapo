#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    // Use shared memory to optimize memory access
    extern __shared__ float shared_mem[];
    float *e_values = shared_mem;
    float *largest_shared = shared_mem + n;

    int i = threadIdx.x;
    float sum = 0;

    // Find the largest element using parallel reduction
    largest_shared[0] = -INFINITY;
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        float val = input[i * stride];
        atomicMax(reinterpret_cast<int*>(&largest_shared[0]), __float_as_int(val));
    }
    __syncthreads();

    // Calculate exp values and sum them
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        e_values[i] = expf(input[i * stride] / temp - largest_shared[0] / temp);
        atomicAdd(&sum, e_values[i]);
    }
    __syncthreads();

    // Normalize exp values
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        output[i * stride] = e_values[i] / sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;
    
    int s = id % spatial;
    id = id / spatial;
    int g = id % groups;
    int b = id / groups;
    int goff = group_offset[g] * spatial;
    int boff = b * stride;

    // Launch softmax_device kernel with shared memory size
    int shared_mem_size = group_size[g] * sizeof(float) + sizeof(float);
    softmax_device<<<1, 256, shared_mem_size>>>(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}