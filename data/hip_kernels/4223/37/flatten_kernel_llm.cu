#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out) {
    // Calculate unique global index for each thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Exit if index 'i' is out of bounds
    if (i >= N) return;

    // Calculate indices based on global index 'i'
    int in_s = i % spatial;
    int in_c = (i / spatial) % layers;
    int b = i / (spatial * layers);

    // Compute input and output indices
    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;
    
    // Assign to output based on the 'forward' flag
    if (forward) out[i2] = x[i1];
    else out[i1] = x[i2];
}