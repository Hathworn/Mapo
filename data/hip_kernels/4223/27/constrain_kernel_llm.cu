#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure no out-of-bounds memory access
    if(i < N)
    {
        // Clamp the value using ALPHA
        int idx = i * INCX;
        float val = X[idx];
        X[idx] = fminf(ALPHA, fmaxf(-ALPHA, val));
    }
}