#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;
    float mean_value = mean[filter]; // Load mean once to reduce global memory access

    for(int j = 0; j < batch; ++j){
        for(int i = id; i < spatial; i += threads){
            int index = j * spatial * filters + filter * spatial + i;
            if(i < spatial) {
                float diff = x[index] - mean_value;
                local[id] += diff * diff; // Use intermediate variable to avoid redundant powf call
            }
        }
    }

    __syncthreads();

    if(id == 0){
        variance[filter] = 0;
        for(int i = 0; i < threads; ++i){
            variance[filter] += local[i];
        }
        variance[filter] /= (spatial * batch - 1);
    }
}