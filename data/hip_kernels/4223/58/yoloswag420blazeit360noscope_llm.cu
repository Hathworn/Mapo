#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation
    int stride = gridDim.x * blockDim.x; // Calculate stride for grid stride loop

    for (int i = id; i < size; i += stride) { // Use grid stride loop for better load balancing
        input[i] = (rand[i] < prob) ? 0 : input[i] * scale; // Update input based on condition
    }
}