#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified id calculation
    if (id >= size) return;

    int temp_id = id;
    int i = temp_id % minw;
    temp_id /= minw;
    int j = temp_id % minh;
    temp_id /= minh;
    int k = temp_id % minc;
    temp_id /= minc;
    int b = temp_id % batch;

    // Combined index calculations
    int out_index = i*sample + w2*(j*sample + h2*(k + c2*b));
    int add_index = i*stride + w1*(j*stride + h1*(k + c1*b));

    // Optimize computation by using fma if possible
    out[out_index] = __fmaf_rn(s1, out[out_index], s2 * add[add_index]);
}