#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -(size / 2);
    int h_offset = -(size / 2);

    int out_index = j + w * (i + h * (k + c * b));
    float out_value = x[out_index];  // Cache x[out_index] for reuse

    for (int l = 0; l < size; ++l) {
        int cur_h = h_offset + i + l;
        if (cur_h < 0 || cur_h >= h) continue;  // Skip invalid rows
        for (int m = 0; m < size; ++m) {
            int cur_w = w_offset + j + m;
            if (cur_w < 0 || cur_w >= w) continue;  // Skip invalid columns
            int index = cur_w + w * (cur_h + h * (k + b * c));
            delta[out_index] += rate * (x[index] - out_value);
        }
    }
}