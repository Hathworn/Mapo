#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    // Calculate global index
    int s = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    
    // Early exit if the index is out of bounds
    if (s >= size) return;

    // Use shared memory to store the mean
    __shared__ float sharedMean;
    
    // Calculate mean using parallel reduction
    float localMean = 0;
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        localMean += fabsf(input[i * size + s]);
    }
    
    // Sum local mean values into shared memory
    atomicAdd(&sharedMean, localMean / n);
    __syncthreads();
    
    // Binarize input using the mean from shared memory
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? sharedMean : -sharedMean;
    }
}