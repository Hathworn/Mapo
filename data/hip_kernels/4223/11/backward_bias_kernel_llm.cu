#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Unroll the inner loop to enhance performance
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Use a reduction pattern to sum partial results more efficiently
    if (p == 0) {
        float blockSum = 0;
        for (int i = 0; i < BLOCK; ++i) {
            blockSum += part[i];
        }
        atomicAdd(&bias_updates[filter], blockSum); // Use atomic operation to prevent race conditions
    }
}