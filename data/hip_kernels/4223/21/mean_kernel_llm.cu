#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Pre-compute scale and setup thread index
    float scale = 1.f / (batch * spatial);
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    if (i >= filters) return;
    
    // Initialize mean accumulator
    float sum = 0;
    
    // Perform computation, using shared memory could be beneficial if needed for large data
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += x[index];
        }
    }
    
    // Finalize mean calculation
    mean[i] = sum * scale;
}