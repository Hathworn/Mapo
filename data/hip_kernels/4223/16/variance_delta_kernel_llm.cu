#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Calculate the index of the current thread in the grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure the thread index is within bounds
    if (i >= filters) return;
    float temp_sum = 0.0f;

    // Compute indices in a loop to accumulate the sum
    for (int j = 0; j < batch; ++j){
        for (int k = 0; k < spatial; ++k){
            int index = j * filters * spatial + i * spatial + k;
            temp_sum += delta[index] * (x[index] - mean[i]);
        }
    }
    // Do final computation on temp_sum
    variance_delta[i] = temp_sum * -.5f * powf(variance[i] + .00001f, -1.5f);
}