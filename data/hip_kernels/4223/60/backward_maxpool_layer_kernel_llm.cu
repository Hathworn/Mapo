#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    // Calculate output spatial dimensions
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int area = (size - 1) / stride;

    // Calculate the unique global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) return;

    // Compute initial positions and offsets
    int j = index % in_w;
    int i = (index / in_w) % in_h;
    int k = (index / (in_w * in_h)) % in_c;
    int b = index / (in_w * in_h * in_c);

    int w_offset = -pad;
    int h_offset = -pad;

    float d = 0;
    
    // Precompute stride-based output position for the center element
    int center_out_w = (j - w_offset) / stride;
    int center_out_h = (i - h_offset) / stride;

    // Loop through the receptive field
    for (int l = -area; l <= area; ++l) {
        for (int m = -area; m <= area; ++m) {
            int out_w = center_out_w + m;
            int out_h = center_out_h + l;

            // Check valid output positions
            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h) {
                int out_index = out_w + w * (out_h + h * (k + c * b));
                if (indexes[out_index] == index) {
                    // Aggregate contributions from delta if the index matches
                    d += delta[out_index];
                }
            }
        }
    }

    // Update the prev_delta
    prev_delta[index] += d;
}