#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan_y(int *g_odata, int *g_idata, int n)
{
    extern __shared__ int temp[];
    int thid = threadIdx.x;
    int bid = blockIdx.x;
    int bdim = blockDim.x;
    int gdim = gridDim.x;
    int offset = 1;
    
    // Load input into shared memory with coalesced access
    int index = bid * bdim * 2 + thid;
    temp[2*thid]   = g_idata[index];
    temp[2*thid+1] = g_idata[index + bdim];
    
    // Reduction phase to build the sum
    for (int d = n>>1; d > 0; d >>= 1)
    {
        __syncthreads();
        if (thid < d)
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }
    
    // Clear the last element
    if (thid == 0) { temp[n - 1] = 0; }
    
    // Down-sweep phase to build the scan
    for (int d = 1; d < n; d *= 2)
    {
        offset >>= 1;
        __syncthreads();
        if (thid < d)
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();
    
    // Write results to device memory
    g_odata[index] = temp[2*thid+1];
    if (2*thid+2 < n)
    {
        g_odata[index + bdim] = temp[2*thid+2];
    }
    else
    {
        g_odata[index + bdim] = temp[2*thid+1] + g_idata[index + bdim];
    }
}