#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global index directly
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use stride loop to handle larger arrays efficiently
    for (; i < N; i += stride) {
        Y[i * INCY] *= X[i * INCX];
    }
}