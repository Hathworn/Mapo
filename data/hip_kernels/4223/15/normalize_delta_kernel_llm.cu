#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds.
    if (index < N) {
        int f = (index / spatial) % filters;

        // Pre-compute values that are used multiple times.
        float inv_sqrt_var = rsqrtf(variance[f] + 0.00001f);
        float common_factor = 1.0f / (spatial * batch);

        // Optimize the delta update calculation
        delta[index] = delta[index] * inv_sqrt_var 
                      + variance_delta[f] * 2.0f * (x[index] - mean[f]) * common_factor 
                      + mean_delta[f] * common_factor;
    }
}