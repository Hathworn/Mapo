#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation

    if (i < n) {
        float t = truth[i];
        float p = pred[i];
        float log_p_safe = logf(p + 1e-7f); // Use logf and more consistent small constant
        float log_1mp_safe = logf(1.0f - p + 1e-7f); // Use logf and more consistent small constant
        error[i] = -t * log_p_safe - (1.0f - t) * log_1mp_safe;
        delta[i] = t - p;
    }
}