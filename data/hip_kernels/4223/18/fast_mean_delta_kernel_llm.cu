#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    
    // Initialize shared memory to zero
    local[id] = 0;

    // Accumulate delta values for each thread
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {  // Use thread stride
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += (i < spatial) ? delta[index] : 0;
        }
    }

    __syncthreads();

    // Reduce sum in a single thread
    if (id == 0) {
        float sum = 0;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        mean_delta[filter] = sum * (-1.f / sqrtf(variance[filter] + .00001f));
    }
}