#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1.f / (batch * spatial - 1);
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Initialize variance
    float var = 0.0f;
    
    // Loop over batch and spatial dimensions, computing variance
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            float diff = x[index] - mean[i];
            var += diff * diff; // Avoid using powf for better performance
        }
    }
    
    // Store the computed variance with scaling
    variance[i] = var * scale;
}