#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    int stride = gridDim.x * blockDim.x;  // Calculate total number of threads

    for (int i = index; i < N; i += stride) {  // Iterate over elements in increments of total threads
        int f = (i / spatial) % filters;
        x[i] = (x[i] - mean[f]) / (sqrtf(variance[f] + .00001f));
    }
}