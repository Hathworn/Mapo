#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void make_pillar_index_kernel(int* dev_pillar_count_histo, int* dev_counter, int* dev_pillar_count, int* dev_x_coors, int* dev_y_coors, float* dev_x_coors_for_sub, float* dev_y_coors_for_sub, float* dev_num_points_per_pillar, int* dev_sparse_pillar_map, const int max_pillars, const int max_points_per_pillar, const int GRID_X_SIZE, const float PILLAR_X_SIZE, const float PILLAR_Y_SIZE, const int NUM_INDS_FOR_SCAN)
{
    // Calculate indexes for current thread
    int x = blockIdx.x;
    int y = threadIdx.x;
    
    // Calculate number of points in this pillar
    int num_points_at_this_pillar = dev_pillar_count_histo[y * GRID_X_SIZE + x];
    if (num_points_at_this_pillar == 0)
    {
        return;  // Early exit for empty pillars
    }

    int count = atomicAdd(dev_counter, 1);
    if (count < max_pillars)
    {
        atomicAdd(dev_pillar_count, 1);
        
        // Populate point count up to max_points_per_pillar
        dev_num_points_per_pillar[count] = min(num_points_at_this_pillar, max_points_per_pillar);
        
        // Assign coordinates
        dev_x_coors[count] = x;
        dev_y_coors[count] = y;
        
        // Calculate offsets for sub-coordinates
        dev_x_coors_for_sub[count] = x * PILLAR_X_SIZE + 0.1f;
        dev_y_coors_for_sub[count] = y * PILLAR_Y_SIZE - 39.9f;
        
        // Mark the pillar as occupied
        dev_sparse_pillar_map[y * NUM_INDS_FOR_SCAN + x] = 1;
    }
}