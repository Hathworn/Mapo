#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = blockDim.x * gridDim.x;
    
    for (; i < n; i += gridStride) { // Use grid-stride loop for better occupancy
        float diff = truth[i] - pred[i];
        error[i] = diff * diff;       // Compute squared error
        delta[i] = diff;              // Compute delta
    }
}