#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int gridSize = blockDim.x * gridDim.x; // Calculate grid stride

    // Loop through the array with a grid stride loop
    while (i < N) {
        float value = X[i * INCX]; // Cache the value to reduce repetitive memory access
        if ((value * value) < (ALPHA * ALPHA)) {
            X[i * INCX] = 0; // Set the element to zero if the condition is met
        }
        i += gridSize; // Move to the next element in grid stride
    }
}