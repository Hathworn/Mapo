#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    // Calculate 1.f - powf(B1, t) and 1.f - powf(B2, t) outside the loop
    // for performance improvement
    float B1_t = 1.f - powf(B1, t);
    float B2_t = 1.f - powf(B2, t);

    int index = blockIdx.x * blockDim.x + threadIdx.x; // More efficient index calculation
    if (index >= N) return;

    float mhat = m[index] / B1_t;
    float vhat = v[index] / B2_t;

    x[index] += rate * mhat / (sqrtf(vhat) + eps); // Simplified update operation
}