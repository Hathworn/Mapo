#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void wgan_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify index calculation
    if(i < n) {
        float t = truth[i];  // Load truth[i] once for efficiency
        error[i] = t ? -pred[i] : pred[i];  // Direct conditional assignment
        delta[i] = (t > 0) ? 1.0f : -1.0f;  // Use float values for delta
    }
}