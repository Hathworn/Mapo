#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Early return if 'offset' is out of bounds
    if(offset >= size) return;

    // Inline computation for better performance
    float bias = biases[filter];
    output[(batch * n + filter) * size + offset] *= bias;
}