#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Calculate the global thread index using a single multiplication
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check the condition and assign values in one line
    if (i < n) binary[i] = (x[i] >= 0) ? 1 : -1;
}