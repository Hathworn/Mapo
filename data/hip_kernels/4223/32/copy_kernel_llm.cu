#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Optimize calculation of index using blockIdx.x and blockIdx.y 
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundary condition and copy if within limits
    if(globalIdx < N) 
    {
        // Optimized memory access pattern
        Y[globalIdx * INCY + OFFY] = X[globalIdx * INCX + OFFX];
    }
}