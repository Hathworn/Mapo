#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    int b = id / c; // Combine operations for better readability and performance

    int out_index = k + c * b;
    float out_val = out_delta[out_index] / (w * h); // Precompute accumulated value
    int base_in_index = h * w * (k + b * c); // Precompute base index

    for(int i = 0; i < w * h; ++i) {
        int in_index = i + base_in_index;
        in_delta[in_index] += out_val; // Use precomputed value to reduce computation
    }
}