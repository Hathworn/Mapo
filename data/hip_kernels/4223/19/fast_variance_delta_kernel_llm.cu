#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta) 
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    int spatial_offset = blockDim.x * gridDim.x;

    local[id] = 0.0f;

    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += spatial_offset) {
            int index = j * spatial * filters + filter * spatial + i;
            // Accumulate delta
            local[id] += delta[index] * (x[index] - mean[filter]);
        }
    }

    __syncthreads();

    // Reduction step
    for (int stride = threads / 2; stride > 0; stride >>= 1) {
        if (id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    // Compute variance delta for this filter
    if (id == 0) {
        variance_delta[filter] = local[0] * (-0.5f) * powf(variance[filter] + 0.00001f, -1.5f);
    }
}