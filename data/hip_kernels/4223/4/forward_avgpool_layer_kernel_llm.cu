#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for average pooling
__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified block index calculation
    if (id >= n) return;

    int k = id % c;
    int b = id / c; // Combined division and assignment

    int out_index = k + c * b;
    float sum = 0.0f; // Use float variable for better precision

    for (int i = 0; i < w * h; ++i) {
        int in_index = i + h * w * (k + b * c);
        sum += input[in_index]; // Accumulate into local variable
    }

    output[out_index] = sum / (w * h); // Directly compute the average with local variable
}