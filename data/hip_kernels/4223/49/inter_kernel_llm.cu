#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute total number of elements per batch
    int total_elements_per_batch = NX + NY;

    // Loop over batches
    for (int i = index; i < (NX + NY) * B; i += blockDim.x * gridDim.x) {
        int b = i / total_elements_per_batch; // Determine batch index
        int j = i % total_elements_per_batch; // Determine element index within the batch

        // Select element from X or Y based on index
        if (j < NX) {
            OUT[i] = X[b * NX + j];
        } else {
            OUT[i] = Y[b * NY + j - NX];
        }
    }
}