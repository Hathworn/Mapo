#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;  // Calculate the stride for the loop

    // Loop through the elements with a stride equal to the total number of threads
    for(; i < N; i += stride) {
        X[i * INCX] *= ALPHA;
    }
}