#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX) {
    // Calculate the global index for the element
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop to handle more elements per thread
    int stride = gridDim.x * blockDim.x;
    for (int index = i; index < N; index += stride) {
        X[index * INCX] = ALPHA;
    }
}