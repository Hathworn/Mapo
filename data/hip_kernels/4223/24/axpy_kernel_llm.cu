#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate thread index using 2D grid and 3D block (more flexibility in block configuration)
    int i = (blockIdx.x * blockDim.x + threadIdx.x) + (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x;

    // Ensure current thread index is within bounds
    if (i < N) {
        // Perform the axpy operation
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}