#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (index >= n) return;

    int b;
    float sum = 0.0f;

    for(b = 0; b < batch; ++b) {
        int i = b * n + index;
        sum += delta[i];
    }

    bias_updates[index] += sum;
}