#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop over elements with stride for better memory access pattern
    for (; i < n; i += stride) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);

        if (abs_val < 1) {
            error[i] = diff * diff;
            delta[i] = diff;
        } else {
            error[i] = 2 * abs_val - 1;
            delta[i] = (diff > 0) ? 1 : -1;
        }
    }
}