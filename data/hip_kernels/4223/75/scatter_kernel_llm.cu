#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scatter_kernel(int *x_coors, int *y_coors, float *pfe_output, float *scattered_feature, const int MAX_NUM_PILLARS_, const int GRID_X_SIZE, const int GRID_Y_SIZE)
{
    // Use shared memory to cache frequently accessed values
    __shared__ int x_ind_shared;
    __shared__ int y_ind_shared;

    if (threadIdx.x == 0) {
        x_ind_shared = x_coors[blockIdx.x];
        y_ind_shared = y_coors[blockIdx.x];
    }
    __syncthreads();

    // Calculate global feature index using shared memory variables
    int i_feature = threadIdx.x;
    float feature = pfe_output[i_feature * MAX_NUM_PILLARS_ + blockIdx.x];
    int global_index = i_feature * GRID_Y_SIZE * GRID_X_SIZE + y_ind_shared * GRID_X_SIZE + x_ind_shared;
    scattered_feature[global_index] = feature;
}