#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    int f1 = index / n;
    int f2 = index % n;
    
    // Use shared memory for faster access
    extern __shared__ float shared_output[];
    
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;

    // Compute dot product and norms using shared memory
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            
            shared_output[threadIdx.x] = output[i1];
            __syncthreads();
            sum += shared_output[threadIdx.x] * output[i2];
            norm1 += shared_output[threadIdx.x] * shared_output[threadIdx.x];
            // Assume output[i2] is already in shared memory
            norm2 += output[i2] * output[i2];
            __syncthreads();
        }
    }

    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;

    sum = sum / norm;

    // Update delta using computed sum
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            delta[i1] += -scale * sum * output[i2] / norm;
            delta[i2] += -scale * sum * output[i1] / norm;
        }
    }
}