#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate index based on 1D block
    int gridSize = blockDim.x * gridDim.x; // Grid stride loop
    while (i < n) {
        if (mask[i] == mask_num) x[i] = val; // Check condition and set value
        i += gridSize; // Move to the next index in the grid stride
    }
}