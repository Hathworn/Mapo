#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Pre-compute shared variables
    extern __shared__ float shared[];
    float* shared_weights = shared;

    int i = threadIdx.x;
    float mean = 0;

    // Parallel mean calculation using shared memory
    for (int stride = 0; stride < size; stride += blockDim.x) {
        if (stride + i < size) {
            shared_weights[i] = fabsf(weights[f * size + stride + i]);
        }
        __syncthreads();

        for (int j = blockDim.x / 2; j > 0; j /= 2) {
            if (i < j) {
                shared_weights[i] += shared_weights[i + j];
            }
            __syncthreads();
        }
        
        if (i == 0) {
            mean += shared_weights[0];
        }
        __syncthreads();
    }

    // Normalize mean
    if (i == 0) mean /= size;
    __syncthreads();

    // Binarize weights
    if (i < size) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    }
}