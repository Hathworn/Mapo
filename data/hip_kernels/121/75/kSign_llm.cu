#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSign(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        target[idx] = copysignf(1.0f, mat[idx]);  // Simplified conditional and improved parallelism
    }
}