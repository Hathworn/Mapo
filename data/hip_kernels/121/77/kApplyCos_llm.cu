#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplyCos(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Simplified loop to handle each element within thread's range.
    if (idx < len) target[idx] = __cosf(mat[idx]);
}