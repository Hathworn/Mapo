#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kRectifiedLinearDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate unique thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop for better memory coalescing
    for (unsigned int i = idx; i < numEls; i += blockDim.x * gridDim.x) {
        dest[i] = a[i] * (b[i] > 0 ? 1 : 0);  // Compute ReLU derivative
    }
}