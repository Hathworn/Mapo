#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSumRowwise(float* mat, float* target, unsigned int width, unsigned int height, float mult, float p) {
    extern __shared__ float sum_vals[];
    const int row = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    // Ensure the row is within bounds
    if (row < height) {
        float sum = 0.0f;
        // Load row base address
        float *data = mat + row;

        // Loop unrolling for improved performance
        for (unsigned int i = 0; i < width; i+=4) {
            if (i < width) sum += data[i * height];
            if (i+1 < width) sum += data[(i+1) * height];
            if (i+2 < width) sum += data[(i+2) * height];
            if (i+3 < width) sum += data[(i+3) * height];
        }

        // Wait for all threads to synchronize (not actually needed here since we are working on independent rows)
        __syncthreads();

        // Write the result to the target
        target[row] = p * target[row] + mult * sum;
    }
}