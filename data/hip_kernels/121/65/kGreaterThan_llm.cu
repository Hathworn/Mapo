#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kGreaterThan(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) { // Only process if idx is within bounds to avoid excess computation
        target[idx] = mat1[idx] > mat2[idx];
    }
}