#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kRMSProp(float *history, float *grad, float factor, int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    for (unsigned int i = idx; i < len; i += numThreads) {
        // Optimize repetitive calculations by precomputing reused terms
        float history_i = history[i];
        float grad_i = grad[i];
        float updated_value = factor * history_i * history_i + (1 - factor) * grad_i * grad_i;
        history[i] = sqrt(updated_value);
    }
}