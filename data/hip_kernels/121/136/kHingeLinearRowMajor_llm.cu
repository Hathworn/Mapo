#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kHingeLinearRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float margin) {
    int image_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (image_id < height) {
        mat += image_id;
        target += image_id;
        const int correct_label = (int)labels[image_id];
        const float correct_label_score = mat[correct_label * height];
        float sum = 0;

        // Use shared memory to reduce repeated global memory fetch
        extern __shared__ float shared_mat[];
        if (threadIdx.x < width) {
            shared_mat[threadIdx.x] = mat[threadIdx.x * height];
        }
        __syncthreads();

        for (unsigned int i = 0; i < width; i++) {
            float diff = margin + shared_mat[i] - correct_label_score; // Read from shared memory
            float grad = (diff > 0) ? 1 : 0;
            target[i * height] = (i == correct_label) ? 0 : grad;
            sum += (i == correct_label) ? 0 : grad;
        }
        target[correct_label * height] = -sum;
    }
}