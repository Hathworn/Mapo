#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLessThanScalar(float* mat, float val, float* target, unsigned int len) {
    // Calculate the global index using efficient built-in functions
    unsigned int idx = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;

    // Check bounds before performing the operation
    if (idx < len) {
        target[idx] = mat[idx] < val;
    }
}