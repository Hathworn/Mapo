#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddMultSign(float* a, float* b, unsigned int numEls, float mult) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += stride) {
        float b_val = b[i]; // Load b[i] into a register
        float mult_sign = (b_val > 0) ? mult : ((b_val < 0) ? -mult : 0);
        a[i] += mult_sign; // Directly add to a[i] to avoid extra computation
    }
}