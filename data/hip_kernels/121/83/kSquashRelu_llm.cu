#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSquashRelu(float* mat, float* target, unsigned int len, float lambda) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) { // Check if idx is within array bounds
        target[idx] = 2 / (1 + __expf(-lambda * mat[idx])) - 1; // Compute target using ReLU formula
    }
}