#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddToEachPixel(float* mat1, float* mat2, float* tgtMat, float mult, unsigned int width, unsigned int height, unsigned int num_pix) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Cache the height to reduce redundant calculations within the loop
    const unsigned int heightMult = height * num_pix;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        // Optimize index calculation by avoiding repeated modulo and division operations
        unsigned int row = i / heightMult;
        unsigned int col = i % height;
        tgtMat[i] = mat1[i] + mult * mat2[col + height * row];
    }
}