#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLog(float* mat, float* target, unsigned int len, float tiny) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    // Use loop unrolling for optimization
    for (; idx + 3 < len; idx += stride * 4) {
        target[idx] = __logf(mat[idx] + tiny);
        target[idx + 1] = __logf(mat[idx + 1] + tiny);
        target[idx + 2] = __logf(mat[idx + 2] + tiny);
        target[idx + 3] = __logf(mat[idx + 3] + tiny);
    }
    for (; idx < len; idx += stride) {
        target[idx] = __logf(mat[idx] + tiny);
    }
}