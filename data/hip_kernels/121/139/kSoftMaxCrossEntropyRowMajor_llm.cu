#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSoftMaxCrossEntropyRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float tiny) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Iterate over rows this thread is responsible for
    for (unsigned int i = idx; i < height; i += stride) {
        int label_idx = height * static_cast<int>(labels[i]) + i;
        float mat_val = mat[label_idx];
        
        // Efficient calculation with prefetching the matrix value
        target[i] = -__logf(mat_val + tiny);
    }
}