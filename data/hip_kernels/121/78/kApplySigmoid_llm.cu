#include "hip/hip_runtime.h"
```cuda
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float sigmoid(float x) {
    return 1.0f / (1.0f + __expf(-x));
}

__global__ void kApplySigmoid(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) { // Ensure idx is within bounds
        target[idx] = sigmoid(mat[idx]);
    }
}