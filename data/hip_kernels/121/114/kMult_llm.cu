#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMult(float* a, float* b, float* dest, unsigned int numEls, float scale_targets) {
    // Use shared memory to reduce global memory access latency
    extern __shared__ float sharedMem[];

    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int tid = threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Copy data to shared memory
    if (idx < numEls) {
        sharedMem[tid] = a[idx] * b[idx];
    }
    __syncthreads();

    if (scale_targets == 0) {
        for (unsigned int i = idx; i < numEls; i += numThreads) {
            dest[i] = sharedMem[tid];
        }
    } else {
        for (unsigned int i = idx; i < numEls; i += numThreads) {
            dest[i] = scale_targets * dest[i] + sharedMem[tid];
        }
    }
}