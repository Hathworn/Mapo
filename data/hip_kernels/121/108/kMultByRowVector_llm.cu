#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate row and column index
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Use row-major order to access elements based on row and column indices
    if (row < height && col < width) {
        unsigned int idx = row * width + col;
        tgtMat[idx] = mat[idx] * vec[row];
    }
}