#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLowerBound(float* mat1, float* mat2, float* target, unsigned int len) {
    // Calculate flattened grid index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize loop by checking bounds inside loop
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        target[i] = fmaxf(mat1[i], mat2[i]); // Use fmaxf for clarity and potential performance benefits
    }
}