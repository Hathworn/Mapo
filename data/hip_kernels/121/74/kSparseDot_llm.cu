#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSparseDot(int m, int n, int k, float *data, int* indptr, int* indices, float *dense_data, float* target, float beta, float alpha) {
    // Calculate row and column indices for threads
    const unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds for row and column indices
    if (row < m && col < n) {
        // Access start and end indices for sparse matrix's current row
        const int start = indptr[row];
        const int end = indptr[row + 1];

        float sum = 0.f;

        // Unroll loop to improve performance
        for (int i = start; i < end; i++) {
            sum += data[i] * dense_data[col * k + indices[i]];
        }

        // Compute position in target array
        const int pos = col * m + row;

        // Update target array
        target[pos] = alpha * sum + ((beta == 0) ? 0 : beta * target[pos]);
    }
}