#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel with coalesced memory access and reduced register pressure
__global__ void kAddScalar(float* a, float alpha, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEls) {
        dest[idx] = a[idx] + alpha; // Direct access for better memory coalescing
    }
}