#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kUpperBound(float* mat1, float* mat2, float* target, unsigned int len) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize loop by ensuring each thread processes contiguous memory
    // Accessing contiguous memory helps with coalesced memory accesses
    if (idx < len) {
        target[idx] = mat1[idx] > mat2[idx] ? mat2[idx] : mat1[idx];
    }
}