#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTanhDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
   
    // Ensure the thread operates only within bounds
    if (idx < numEls) {
        dest[idx] = a[idx] * (1.0 + b[idx]) * (1.0 - b[idx]);
    }
}