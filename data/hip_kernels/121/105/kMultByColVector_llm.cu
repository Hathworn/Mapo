#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kMultByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate linear index based on block and thread indices
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x; // Total number of threads

    // Iterate over matrix elements in a strided pattern
    for (unsigned int i = idx; i < width * height; i += stride) {
        unsigned int row = i % height; // Determine the current row
        tgtMat[i] = mat[i] * vec[row]; // Multiply the matrix element by corresponding vector element
    }
}