#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddDiagonal(float* mat, float* vec, float* tgtMat, unsigned int width) {
    // Calculate unique index for this thread
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Process diagonal elements only if within matrix bounds
    if (idx < width) {
        tgtMat[width * idx + idx] = mat[width * idx + idx] + vec[idx];
    }
}