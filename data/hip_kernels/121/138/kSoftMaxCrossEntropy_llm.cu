#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSoftMaxCrossEntropy(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float tiny) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int gridStride = blockDim.x * gridDim.x;
    
    // Loop with grid-stride to efficiently utilize all threads
    for (unsigned int i = idx; i < width; i += gridStride) {
        int labelIdx = height * i + static_cast<int>(labels[i]);
        target[i] = -__logf(mat[labelIdx] + tiny); // Ensure label index computation only once
    }
}