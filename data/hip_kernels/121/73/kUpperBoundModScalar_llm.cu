#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kUpperBoundModScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a grid-stride loop for better performance
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        float matVal = mat[i];
        target[i] = matVal > val ? val : (matVal < -val ? -val : matVal);
    }
}