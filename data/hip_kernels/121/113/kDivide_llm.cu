#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivide(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate the thread's global index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride to handle all elements
    for (unsigned int i = idx; i < numEls; i += blockDim.x * gridDim.x) {
        // Perform division if not dividing by zero
        if (b[i] != 0.0f) {
            dest[i] = a[i] / b[i];
        } else {
            dest[i] = 0.0f;  // Handle division by zero
        }
    }
}