#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddColMult(float* mat, float* vec, float* tgtMat, float mult, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    unsigned int i = idx;

    // Loop unrolling for better performance
    while (i + stride < width * height) {
        tgtMat[i] = mat[i] + mult * vec[i % height];
        tgtMat[i + stride] = mat[i + stride] + mult * vec[(i + stride) % height];
        i += 2 * stride;
    }

    if (i < width * height) {
        tgtMat[i] = mat[i] + mult * vec[i % height];
    }
}