#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kPow(float* mat, float pow, float* target, unsigned int len) {
    // Calculate global thread ID
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll loop to enhance performance
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        target[i] = powf(mat[i], pow);
    }
}