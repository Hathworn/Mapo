#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogisticDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEls) { // Check to ensure idx is within bounds
        float b_val = b[idx]; // Cache frequently accessed value
        dest[idx] = a[idx] * b_val * (1.0f - b_val);
    }
}