#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetOnes(float *dest, int count) {
    // Use block and grid indices to ensure unique threads for each element
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count) {
        dest[idx] = 1.0f;
    }
}