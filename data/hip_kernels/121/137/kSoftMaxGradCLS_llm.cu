#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSoftMaxGradCLS(float* mat, int* labels, float* indices, float* target, unsigned int width, unsigned int height) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory for frequently accessed data
    __shared__ int shared_labels[1024];  // Adjust size if needed

    // Load labels into shared memory
    if (threadIdx.x < height) {
        shared_labels[threadIdx.x] = labels[threadIdx.x];
    }
    __syncthreads();

    // Unroll loop for better performance
    for (unsigned int i = idx; i < width * height; i += blockDim.x * gridDim.x) {
        // Compute target using shared memory for labels
        target[i] = mat[i] - (shared_labels[(int)indices[i % height]] == i / height ? 1.0f : 0.0f);
    }
}