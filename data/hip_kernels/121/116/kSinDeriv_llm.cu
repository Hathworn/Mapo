#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSinDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate unique thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Iterate over elements this thread should handle
    if (idx < numEls) {
        dest[idx] = a[idx] * __cosf(b[idx]); // Compute result
    }
}
```
