#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCeil(float* mat, float* target, unsigned int len) {
    // Calculate the global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if index is within array bounds
    if (idx < len) {
        target[idx] = ceil(mat[idx]);
    }
}