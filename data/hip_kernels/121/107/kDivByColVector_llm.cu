#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate 2D grid and block indices for improved memory coalescing
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds to prevent accessing out of range
    if (col < width && row < height) {
        unsigned int idx = row * width + col;
        tgtMat[idx] = mat[idx] / vec[row];
    }
}