#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void kOutpTraceMultiplyImages(float *expanded_images, float *expanded_derivs, int num_images, int num_channels, int num_modules_batch, int kernel_size) {
    int color = blockIdx.y;
    int module_id = blockIdx.x;
    int thread_id = threadIdx.x;

    // Compute offsets outside of the loop to avoid repeated calculations
    float *images_base = expanded_images + num_images * (num_modules_batch * kernel_size * color + module_id);
    float *derivs_base = expanded_derivs + num_images * (num_modules_batch * color + module_id);

    for (int kpos = 0; kpos < kernel_size; kpos++) {
        for (int im = thread_id; im < num_images; im += blockDim.x) {
            // Use precomputed base pointers
            int image_idx = im + num_images * num_modules_batch * kpos;
            int deriv_idx = im;
            images_base[image_idx] *= derivs_base[deriv_idx];
        }
        
        __syncthreads(); // Ensure all threads have completed before the next iteration
    }
}