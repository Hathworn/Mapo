#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kGreaterThanScalar(float* mat, float val, float* target, unsigned int len) {
    // Calculate the global index of the thread
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure that only valid indices are processed
    if (idx < len) {
        target[idx] = mat[idx] > val;
    }
}