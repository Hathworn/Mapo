#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel to utilize memory coalescing and reduce loop overhead
__global__ void kApplySin(float* __restrict__ mat, float* __restrict__ target, unsigned int len) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {  // Ensure each thread accesses valid memory
        target[idx] = __sinf(mat[idx]);  // Directly calculate sine for each thread index
    }
}