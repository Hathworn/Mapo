#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplyTanh(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Use register for frequently accessed variables.
    float mat_i, exp2x;

    for (unsigned int i = idx; i < len; i += numThreads) {
        mat_i = mat[i];
        exp2x = __expf(2.0f * mat_i);  // Ensure floating point precision.
        target[i] = 1.0f - 2.0f / (exp2x + 1.0f);  // Use 1.0f for consistency in floats.
    }
}