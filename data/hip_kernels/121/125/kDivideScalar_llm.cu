#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivideScalar(float* mat, float alpha, float* dest, unsigned int len) {
    // Use built-in variable to compute index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Loop with stride equal to total number of threads
    while (idx < len) {
        dest[idx] = mat[idx] / alpha;
        idx += blockDim.x * gridDim.x;
    }
}