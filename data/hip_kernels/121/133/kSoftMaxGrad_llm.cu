#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSoftMaxGrad(float* mat, float* labels, float* target, unsigned int width, unsigned int height) {
    // Calculate the unique global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x; // Calculate the stride

    for (unsigned int i = idx; i < width * height; i += stride) { // Use stride in the loop
        unsigned int row = i / height; // Pre-compute row index for labels access
        unsigned int col = i % height; // Pre-compute column index
        target[i] = mat[i] - ((int)labels[row] == col ? 1 : 0); // Simplify condition checking
    }
}