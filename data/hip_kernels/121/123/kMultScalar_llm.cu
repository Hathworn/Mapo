#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultScalar(float* mat, float alpha, float* dest, unsigned int len, float scale_targets) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x; // Use stride for better readability

    // Unroll the loop for potential performance improvement
    if (scale_targets == 0) {
        for (unsigned int i = idx; i < len; i += stride) {
            dest[i] = alpha * mat[i];
        }
    } else {
        for (unsigned int i = idx; i < len; i += stride) {
            dest[i] = scale_targets * dest[i] + alpha * mat[i];
        }
    }
}