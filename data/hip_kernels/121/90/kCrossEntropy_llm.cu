#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCrossEntropy(float* mat, float* p, float* target, unsigned int len, float tiny) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use grid-stride loop for better performance
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        target[i] = -mat[i] * __logf(p[i] + tiny);
    }
}