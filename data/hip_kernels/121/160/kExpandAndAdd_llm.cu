#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kExpandAndAdd(float* source, float* mat, float* indices, float* target, int width, int height, float mult, int width2) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Optimize by reducing repetitive calculations
    for (unsigned int i = idx; i < width * height; i += numThreads) {
        const int row = i / height;
        const int column = i % height;
        const int pos = height * static_cast<int>(indices[row]) + column;
        
        // Simplify target assignment using ternary operator
        target[i] = (pos < height * width2) ? (source[i] + mult * mat[pos]) : NAN;
    }
}