#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kGreaterThanEqScalar(float* mat, float val, float* target, unsigned int len) {
    // Calculate a unique index for each thread
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check that the index is within bounds
    if (idx < len) {
        // Perform comparison and write result
        target[idx] = mat[idx] >= val;
    }
}