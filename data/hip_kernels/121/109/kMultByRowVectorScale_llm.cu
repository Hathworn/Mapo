#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByRowVectorScale(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height, float scale_targets) {
    // Calculate the global linear index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate the total number of threads
    const unsigned int numThreads = blockDim.x * gridDim.x;
    
    // Optimize by reducing number of multiplications inside the loop
    for (unsigned int i = idx; i < width * height; i += numThreads) {
        unsigned int rowIdx = i / width; // Calculate row index
        tgtMat[i] = scale_targets * tgtMat[i] + mat[i] * vec[rowIdx];
    }
}