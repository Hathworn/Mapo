#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kPowMatrix(float* mat, float* pow, float* target, unsigned int len) {
    // Thread index calculation
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Step through array using stride pattern
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        target[i] = powf(mat[i], pow[i]);
    }
}