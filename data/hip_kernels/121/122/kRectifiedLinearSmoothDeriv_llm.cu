#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kRectifiedLinearSmoothDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll loop for efficiency
    for (unsigned int i = idx; i < numEls; i += blockDim.x * gridDim.x) {
        dest[i] = a[i] * (1 - __expf(-b[i]));
    }
}