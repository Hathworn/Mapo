#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kReciprocal(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure valid index within array bounds
    if (idx < len) {
        target[idx] = 1.0f / mat[idx];
    }
}