#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSubtract(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate the index based on block and thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a stride to allow each thread to process multiple elements
    const unsigned int stride = blockDim.x * gridDim.x;

    // Loop through elements, processing multiple indices
    for (unsigned int i = idx; i < numEls; i += stride) {
        dest[i] = a[i] - b[i]; // Element-wise subtraction
    }
}