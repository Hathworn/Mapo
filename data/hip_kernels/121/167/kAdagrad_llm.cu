#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdagrad(float *history, float *grad, float delta, int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Ensure index is within bounds and use strided loop
    for (unsigned int i = idx; i < len; i += stride) {
        float curr_norm = history[i] - delta;
        // Reduce redundant computation and combine operations for better ILP
        float grad_val = grad[i];
        history[i] = sqrt(curr_norm * curr_norm + grad_val * grad_val) + delta;
    }
}