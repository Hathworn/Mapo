#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kExpand(float* __restrict__ source, const float* __restrict__ indices, float* __restrict__ target, int height, int width, int target_width) {
    // Calculate global index and number of threads
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Loop through the target elements
    for (unsigned int i = idx; i < target_width * height; i += numThreads) {
        // Calculate position in source array using indices
        const int pos = __fmul_rn(height, __float2int_rn(indices[i / height])) + i % height;

        // Conditional assignment using ternary operator
        target[i] = (pos < height * width) ? source[pos] : __int_as_float(0x7fffffff); // Avoid division by zero for NaN
    }
}