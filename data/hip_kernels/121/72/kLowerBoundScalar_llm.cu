#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLowerBoundScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;  // Reduce repetitive calculation
    
    // Use stride variable for loop increment
    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = mat[i] < val ? val : mat[i];
    }
}