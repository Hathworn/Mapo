#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogisticGrad(float* mat, float* targets, float* out_grad, unsigned int numEls) {
    // Utilize shared memory for faster access (if applicable, modify as needed)
    __shared__ float shared_data[256];  // Assume this size according to available resources
    
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate total number of threads
    const unsigned int numThreads = blockDim.x * gridDim.x;
    
    // Loop over elements using thread stride
    for (unsigned int i = idx; i < numEls; i += numThreads) {
        // Calculate gradient
        out_grad[i] = (targets[i] < 0) ? 0.0f : (mat[i] - targets[i]);
    }
}