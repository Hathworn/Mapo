#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCorrectPreds(float* mat, float* p, float* target, unsigned int len, float cutoff) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize: Ensure that threads exceeding array length do not execute
    if (idx < len) {
        // Perform computation directly without loop
        target[idx] = mat[idx] * (p[idx] >= cutoff) + (1 - mat[idx]) * (p[idx] < cutoff);
    }
}