#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kHingeQuadraticRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float margin) {
    // Use shared memory to cache data for the current image
    extern __shared__ float sdata[];
    int image_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (image_id < height) {
        mat += image_id;
        target += image_id;
        const int correct_label = (int)labels[image_id];
        const float correct_label_score = mat[correct_label * height];

        float sum = 0;
        for (unsigned int i = threadIdx.x; i < width; i += blockDim.x) { // Use all threads in the block
            float diff = margin + mat[i * height] - correct_label_score;
            float grad = (diff > 0) ? diff : 0;
            target[i * height] = (i == correct_label) ? 0 : grad;
            sum += (i == correct_label) ? 0 : grad;
        }

        sdata[threadIdx.x] = sum; // Store partial sum in shared memory
        __syncthreads(); // Ensure all threads have written their partial sums
        
        // Reduce partial sums within block
        for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                sdata[threadIdx.x] += sdata[threadIdx.x + stride];
            }
            __syncthreads();
        }
        
        if (threadIdx.x == 0) {
            target[correct_label * height] = -sdata[0];
        }
    }
}