#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void kLessThanEqScalar(float* __restrict__ mat, float val, float* __restrict__ target, unsigned int len) {
    // Use cache-friendly shared memory to improve performance
    extern __shared__ float shared_mem[];
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    for (unsigned int i = idx; i < len; i += stride) {
        shared_mem[threadIdx.x] = mat[i];
        __syncthreads(); // Ensure all threads have written to shared memory
        target[i] = shared_mem[threadIdx.x] <= val; // Use shared memory result
    }
}