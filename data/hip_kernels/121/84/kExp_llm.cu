#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kExp(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a more efficient loop by checking idx directly.
    if (idx < len) {
        target[idx] = __expf(mat[idx]);
    }
}