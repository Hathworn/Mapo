#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using shared memory
__global__ void kApplyLog1PlusExp(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    __shared__ float sharedMat[256]; // Adjust size according to blockDim.x if necessary

    for (unsigned int i = idx; i < len; i += stride) {
        sharedMat[threadIdx.x] = mat[i]; // Load data into shared memory
        __syncthreads(); // Ensure all threads have loaded their data

        float mat_i = sharedMat[threadIdx.x]; // Read from shared memory
        if (mat_i > 0)
            target[i] = (__logf(1 + __expf(-mat_i)) + mat_i);
        else
            target[i] = __logf(1 + __expf(mat_i));

        __syncthreads(); // Ensure results are computed before next iteration
    }
}