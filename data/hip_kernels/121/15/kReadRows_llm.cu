#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kReadRows(float* data, float* target, int num_images, int num_modules, int num_modules_batch, int module_id_offset) {
    int c = blockIdx.y;
    int src_module_id = module_id_offset + blockIdx.x;
    int dst_module_id = blockIdx.x;
    
    // Pre-calculate common offsets to reduce redundant calculations
    int data_offset = num_images * (src_module_id + c * num_modules);
    int target_offset = num_images * (dst_module_id + c * num_modules_batch);

    // Use shared memory for faster access if beneficial (not used here as no repeated access pattern)
    // __shared__ float data_shared[blockDim.x]; 

    data += data_offset;
    target += target_offset;

    // Use warp-level parallelism to optimize data transfer
    int im = threadIdx.x + blockDim.x * blockIdx.z;
    while (im < num_images) {
        target[im] = data[im];
        im += blockDim.x * gridDim.z;
    }
}