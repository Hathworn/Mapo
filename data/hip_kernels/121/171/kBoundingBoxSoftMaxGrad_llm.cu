#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBoundingBoxSoftMaxGrad(float* mat, int* bbox, int* label, int* seg, float* indices, float* width_offset, float* height_offset, int size, int width, int height, int depth, float scale_width, float scale_height, float* grad) {
    const unsigned int len = width * height * depth * size;
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    if (idx >= len) return;  // Boundary check for early exit

    for (unsigned int i = idx; i < len; i += numThreads) {
        int ind = i;
        int image_id = ind % size; ind /= size;
        float source_x = scale_width * (ind % width); ind /= width;
        float source_y = scale_height * (ind % height); ind /= height;
        int source_depth = ind % depth;
        int src_image_id = static_cast<int>(indices[image_id]);

        int start = seg[src_image_id];
        int end = seg[src_image_id + 1];
        int num_bboxes = 0;
        int num_bboxes_of_this_depth = 0;

        #pragma unroll // Improve loop efficiency
        for (int box_id = start; box_id < end; box_id++) {
            int x1 = bbox[box_id << 2] - width_offset[image_id];
            int y1 = bbox[(box_id << 2) + 1] - height_offset[image_id];
            int x2 = bbox[(box_id << 2) + 2] - width_offset[image_id];
            int y2 = bbox[(box_id << 2) + 3] - height_offset[image_id];
            int inside = (source_x >= x1 && source_x <= x2 && source_y >= y1 && source_y <= y2) ? 1 : 0;
            num_bboxes += inside;
            num_bboxes_of_this_depth += (inside && label[box_id] == source_depth) ? 1 : 0;
        }

        grad[i] = mat[i] - ((num_bboxes > 0) ? ((float)num_bboxes_of_this_depth / num_bboxes) : (source_depth == 0 ? 1 : 0));
    }
}