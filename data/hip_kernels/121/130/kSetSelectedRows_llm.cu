#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetSelectedRows(float* target, float* source, float* indices, int nRowIs, int nCols, int nTargetRows) {
    __shared__ int targetRowIndices[32];
    const int startSourceRowI = blockIdx.x * 32;
    const int tid = threadIdx.x;
    const int localNRowIs = min(32, nRowIs - startSourceRowI);

    // Cooperatively load 32 row indices
    #pragma unroll
    for (int i = tid; i < localNRowIs; i += 32) {
        targetRowIndices[i] = int(indices[startSourceRowI + i]);
        if (targetRowIndices[i] < 0)
            targetRowIndices[i] += nTargetRows;
        if (targetRowIndices[i] < 0 || targetRowIndices[i] >= nTargetRows)
            targetRowIndices[i] = -1;
    }
    __syncthreads();

    // Copy 32 rows
    #pragma unroll
    for (int i = 0; i < localNRowIs; i++) {
        const int sourceRowI = startSourceRowI + i;
        const int targetRowI = targetRowIndices[i];
        if (targetRowI != -1) {  // Only proceed if the index is valid
            for (int colI = tid; colI < nCols; colI += 32) {
                target[targetRowI * nCols + colI] = source[sourceRowI * nCols + colI];
            }
        }
    }
}