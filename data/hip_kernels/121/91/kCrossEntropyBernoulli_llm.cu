#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCrossEntropyBernoulli(float* mat, float* p, float* target, unsigned int len, float tiny) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Loop unrolling for better memory access efficiency
    for (unsigned int i = idx; i < len; i += stride) {
        float mat_val = mat[i];
        float p_val = p[i];
        
        // Store logarithm results to avoid redundant calculations
        float log_p = __logf(p_val + tiny);
        float log_one_minus_p = __logf(1 - p_val + tiny);

        target[i] = -mat_val * log_p - (1 - mat_val) * log_one_minus_p;
    }
}