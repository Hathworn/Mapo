#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLessThan(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {  // Ensure index is within bounds
        target[idx] = mat1[idx] < mat2[idx];
    }
}