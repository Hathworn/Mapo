#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kBlockify(float* source, float* target, int numdims, int blocksize) {
    const unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x; // Use global thread index
    const unsigned int stride = blockDim.x * gridDim.x; // Use grid-stride loop

    for (unsigned int target_ind = idx; target_ind < numdims; target_ind += stride) {
        const int block = target_ind / blocksize;
        target[target_ind] = source[block * blocksize]; // Efficient memory access
    }
}