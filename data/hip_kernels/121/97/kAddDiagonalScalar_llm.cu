#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddDiagonalScalar(float* mat, float val, float* tgtMat, unsigned int width) {
    // Calculate row index for the diagonal element
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure only valid diagonal index threads access the array
    if (i < width) {
        tgtMat[width * i + i] = mat[width * i + i] + val;
    }
}