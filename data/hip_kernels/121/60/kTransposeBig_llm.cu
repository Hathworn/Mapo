#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTransposeBig(float *odata, float *idata, int height, int width) {
    __shared__ float tile[32][33];  // Allocate shared memory for coalesced access

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Transpose logic: load data into shared memory for faster access
    if (x < width && y < height) {
        tile[threadIdx.y][threadIdx.x] = idata[y * width + x];
    }
    __syncthreads();

    // Compute transposed position and ensure within bounds
    x = blockIdx.y * blockDim.y + threadIdx.x;
    y = blockIdx.x * blockDim.x + threadIdx.y;
    if (x < height && y < width) {
        odata[x * width + y] = tile[threadIdx.x][threadIdx.y];
    }
}