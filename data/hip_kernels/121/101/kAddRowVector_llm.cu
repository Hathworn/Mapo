#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int threadID = blockIdx.x * blockDim.x + threadIdx.x; // Compute unique thread ID
    const unsigned int stride = gridDim.x * blockDim.x; // Calculate stride

    // Use stride to update matrix more efficiently, promoting data locality
    for (unsigned int i = threadID; i < width * height; i += stride) {
        unsigned int rowIdx = i / width; // Compute row index
        tgtMat[i] = mat[i] + vec[rowIdx]; // Add corresponding row vector element
    }
}