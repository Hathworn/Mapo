#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSqrt(float* mat, float* target, unsigned int len) {
    // Optimize access pattern with coalesced memory access by using stride based on grid size
    unsigned int stride = gridDim.x * blockDim.x;
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += stride) {
        target[i] = sqrt(mat[i]);
    }
}