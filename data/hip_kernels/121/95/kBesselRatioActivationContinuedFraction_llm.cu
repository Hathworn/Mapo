#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kBesselRatioActivationContinuedFraction(float* mat, float* target, float order, int num_terms, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    unsigned int i = idx;
    while (i < len) {  // Optimize loop using while instead of for
        float k = mat[i];
        float result = 2 * (order + num_terms) / k;
        
        // Use shared memory to store intermediate results (if possible)
        // __shared__ float sharedResult;
        // sharedResult = result;
        
        for(int j = num_terms - 1; j > 0; j--) {
            result = 2 * (order + j) / k + 1 / result;
        }
        target[i] = 1 / result;
        i += numThreads;
    }
}