#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCosDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Loop unrolling for improved performance
    for (unsigned int i = idx; i < numEls; i += blockDim.x * gridDim.x) {
        float temp_a = a[i];
        float temp_b = b[i];
        dest[i] = -temp_a * __sinf(temp_b);
    }
}