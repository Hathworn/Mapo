#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTranspose(float *odata, float *idata, int width, int height) {
    __shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE+1];

    // Calculate the input index
    unsigned int xIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
    unsigned int yIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;

    // Load input element into shared memory
    if((xIndex < width) && (yIndex < height)) {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }

    __syncthreads(); // Ensure all threads have loaded the data

    // Calculate the output index
    xIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
    yIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;

    // Write the transposed element to output
    if((xIndex < height) && (yIndex < width)) {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}