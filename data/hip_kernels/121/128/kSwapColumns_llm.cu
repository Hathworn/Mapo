#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSwapColumns(float* __restrict__ source, float* __restrict__ target, float* __restrict__ indices1, float* __restrict__ indices2, int cols, int width, int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Use registers for temporary values
    unsigned int column, row, source_pos, target_pos;

    for (unsigned int i = idx; i < height * cols; i += numThreads) {
        column = i / height;
        row = i % height;

        // Calculate linear indices for source and target positions
        source_pos = __float2int_rd(height * indices1[column]) + row; // use __float2int_rd to convert float to int
        target_pos = __float2int_rd(height * indices2[column]) + row;

        // Perform the swap
        float temp = source[source_pos];
        source[source_pos] = target[target_pos];
        target[target_pos] = temp;
    }
}