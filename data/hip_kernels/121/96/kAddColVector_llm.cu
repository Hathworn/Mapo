#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate unique thread index
    const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Loop through the matrix elements each thread should process
    for (unsigned int i = tid; i < width * height; i += stride) {
        // Perform vector addition
        tgtMat[i] = mat[i] + vec[i % height];
    }
}