#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultDiagonalScalar(float* mat, float val, float* tgtMat, unsigned int width) {
    // Optimize thread grid and loop iteration
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < width) {
        tgtMat[width * idx + idx] = mat[width * idx + idx] * val;
    }
}