#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kGreaterThanEq(float* mat1, float* mat2, float* target, unsigned int len) {
    // Calculate global index for this thread
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure idx is within bounds to prevent out-of-range accesses
    if (idx < len) {
        target[idx] = mat1[idx] >= mat2[idx];
    }
    // No need for loop, as each thread now processes one element
}