#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBesselRatioActivation(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    // Unrolling loop to improve performance
    for (unsigned int i = idx; i < len; i += stride * 4) {
        if (i < len) {
            float r1 = mat[i];
            target[i] = cyl_bessel_i1f(r1) / cyl_bessel_i0f(r1);
        }
        if (i + stride < len) {
            float r2 = mat[i + stride];
            target[i + stride] = cyl_bessel_i1f(r2) / cyl_bessel_i0f(r2);
        }
        if (i + 2 * stride < len) {
            float r3 = mat[i + 2 * stride];
            target[i + 2 * stride] = cyl_bessel_i1f(r3) / cyl_bessel_i0f(r3);
        }
        if (i + 3 * stride < len) {
            float r4 = mat[i + 3 * stride];
            target[i + 3 * stride] = cyl_bessel_i1f(r4) / cyl_bessel_i0f(r4);
        }
    }
}