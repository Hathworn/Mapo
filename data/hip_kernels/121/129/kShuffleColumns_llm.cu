#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kShuffleColumns(float* source, float* target, float* indices, int width, int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    unsigned int column, row, pos1, pos2;

    // Eliminate redundant float variables temp1, temp2; use direct assignment
    for (unsigned int i = idx; i < height * ((width + 1) / 2); i += numThreads) {
        column = 2 * (i / height);
        row = i % height;
        if (column + 1 >= width) {
            pos1 = height * (int)indices[column] + row;
            target[pos1] = source[pos1];
        } else {
            pos1 = height * (int)indices[column] + row;
            pos2 = height * (int)indices[column + 1] + row;
            target[pos2] = source[pos1];
            target[pos1] = source[pos2];
        }
    }
}