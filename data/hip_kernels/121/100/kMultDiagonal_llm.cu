#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultDiagonal(float* mat, float* vec, float* tgtMat, unsigned int width) {
    // Calculate unique thread index with adjusted grid-stride loop
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < width; i += blockDim.x * gridDim.x) {
        // Perform the diagonal multiplication
        tgtMat[width * i + i] = mat[width * i + i] * vec[i];
    }
}