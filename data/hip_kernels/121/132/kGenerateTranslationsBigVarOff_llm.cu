#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kGenerateTranslationsBigVarOff(float* source, float* target, float* off_x_arr, float* off_y_arr, int source_w, int target_w, int num_channels) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; // Use global thread index for better parallelism
    const unsigned int numThreads = blockDim.x * gridDim.x; // Total number of threads

    int pad = (source_w - target_w) / 2;
    int target_tile_size = target_w * target_w;
    int source_tile_size = source_w * source_w;

    int off_x = off_x_arr[blockIdx.x];
    int off_y = off_y_arr[blockIdx.x];
    int target_off = blockIdx.x * target_tile_size;
    int source_off = blockIdx.x * source_tile_size + (pad + off_x) * source_w + (pad + off_y);

    for (unsigned int target_ind = idx; target_ind < target_tile_size; target_ind += numThreads) {
        int target_x = target_ind / target_w;
        int target_y = target_ind % target_w; // Use modulus operator for clearer computation

        for (unsigned int ch = 0; ch < num_channels; ch++) {
            target[num_channels * (target_off + target_x * target_w + target_y) + ch] = 
            source[num_channels * (source_off + target_x * source_w + target_y) + ch];
        }
    }
}