#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogisticCorrectNormalized(float* mat, float* targets, float* out, unsigned int height, unsigned int width) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index is within bounds
    if (idx < height) {
        float correct = 0;
        float total = 0;
        float p, t;

        // Use shared memory for coalesced memory access
        extern __shared__ float shared[];

        // Load data into shared memory
        for (int i = idx; i < width * height; i += height) {
            shared[threadIdx.x] = mat[i];
            __syncthreads();
            
            p = shared[threadIdx.x];
            t = targets[i];

            correct += (t < 0) ? 0 : (((t >= 0.5 && p >= 0.5) || (t < 0.5 && p < 0.5)) ? 1: 0);
            total += (t < 0) ? 0 : 1;
            __syncthreads();
        }

        // Store the result
        out[idx] = (total > 0) ? (correct / total) : 0;
    }
}