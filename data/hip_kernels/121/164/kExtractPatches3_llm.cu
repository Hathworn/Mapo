#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kExtractPatches3(float* images, float* patches, float* width_offset, float* height_offset, float* flip, int num_images, int img_width, int img_height, int patch_width, int patch_height, int num_colors) {

    // Calculate destination pixel position
    int dest_col = blockIdx.x * blockDim.x + threadIdx.x;
    int dest_row = blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary check for patch dimension
    if (dest_col < patch_width && dest_row < patch_height) {
        
        int num_batches = num_colors * num_images;

        // Iterate over each image batch using grid-stride loop
        for (unsigned int b = blockIdx.z; b < num_batches; b += gridDim.z) {
            int color    = b % num_colors;
            int image_id = b / num_colors;
            
            // Compute source row and column
            int source_row = __float2int_rn(height_offset[image_id]) + dest_row;
            int source_col = __float2int_rn(width_offset[image_id]) + dest_col;
            
            // Handle image flipping
            source_col = (flip[image_id] > 0.5f) ? (img_width - source_col - 1) : source_col;
            
            // Calculate indices in the images and patches arrays
            unsigned long source_index = source_col + img_width * (source_row + img_height * (color + num_colors * image_id));
            unsigned long dest_index = dest_col + patch_width * (dest_row + patch_height * (color + num_colors * image_id));
            
            // Assign patch value
            patches[dest_index] = images[source_index];
        }
    }
}