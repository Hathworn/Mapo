#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int totalElements = width * height; // Precompute total elements

    // Access stride-by-stride to improve memory coalescing
    for (unsigned int i = idx; i < totalElements; i += blockDim.x * gridDim.x) {
        tgtMat[i] = mat[i] / vec[i % width]; // Use modulo for correct indexing
    }
}