#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kRectifyBoundingBox(float* boxes, float* width_offset, float* height_offset, float* flip, int num_images, int patch_width, int patch_height, int num_locs) {

    // Calculate block-wide variables once
    int loc_id_start = blockIdx.x;
    int loc_id_step = gridDim.x;
    int image_id_start = threadIdx.x;
    int image_id_step = blockDim.x;

    for (int loc_id = loc_id_start; loc_id < num_locs; loc_id += loc_id_step) {
        // Precompute index computations
        float *xmin_block = boxes + num_images * loc_id;
        float *ymin_block = boxes + num_images * (loc_id + num_locs);
        float *xmax_block = boxes + num_images * (loc_id + num_locs * 2);
        float *ymax_block = boxes + num_images * (loc_id + num_locs * 3);
        
        for (int image_id = image_id_start; image_id < num_images; image_id += image_id_step) {
            // Use registers for frequently accessed values
            float flip_value = flip[image_id];
            float xmin, xmax, ymin, ymax, wo, ho;

            // Decide xmin and xmax based on flipping
            if (flip_value > 0.5) {
                xmin = 256.0 / patch_width - xmax_block[image_id];
                xmax = 256.0 / patch_width - xmin_block[image_id];
            } else {
                xmin = xmin_block[image_id];
                xmax = xmax_block[image_id];
            }

            // Directly fetch values from memory
            ymin = ymin_block[image_id];
            ymax = ymax_block[image_id];
            wo = width_offset[image_id];
            ho = height_offset[image_id];

            // Update bounding box coordinates efficiently
            xmin_block[image_id] = xmin - wo / patch_width;
            xmax_block[image_id] = xmax - wo / patch_width;
            ymin_block[image_id] = ymin - ho / patch_height;
            ymax_block[image_id] = ymax - ho / patch_height;
        }
    }
}