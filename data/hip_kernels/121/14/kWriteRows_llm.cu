#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kWriteRows(float* data, float* target, int num_images, int num_modules, int num_modules_batch, int module_id_offset, float beta) {
    int c = blockIdx.y;
    int src_module_id = blockIdx.x;
    int dst_module_id = module_id_offset + blockIdx.x;

    // Simplified pointer arithmetic
    float* data_ptr = data + num_images * (src_module_id + c * num_modules_batch);
    float* target_ptr = target + num_images * (dst_module_id + c * num_modules);

    int im = threadIdx.x;
    int stride = blockDim.x;
    // Use loop unrolling for the iteration
    #pragma unroll
    for (; im < num_images; im += stride) {
        target_ptr[im] = beta * data_ptr[im];
    }
}