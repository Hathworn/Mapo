#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplyAbs(float* mat, float* target, unsigned int len) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    // Loop unrolling for better performance
    for (unsigned int i = idx; i < len; i += stride * 4) { 
        if (i < len) target[i] = fabsf(mat[i]);
        if (i + stride < len) target[i + stride] = fabsf(mat[i + stride]);
        if (i + 2 * stride < len) target[i + 2 * stride] = fabsf(mat[i + 2 * stride]);
        if (i + 3 * stride < len) target[i + 3 * stride] = fabsf(mat[i + 3 * stride]);
    }
}