#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSoftMaxGradRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Loop with stride to cover all elements efficiently
    for (unsigned int i = idx; i < width * height; i += stride) {
        unsigned int row = i / width;  // Calculate row and column only once
        unsigned int col = i % width;
        target[row * width + col] = mat[row * width + col] - ((int)labels[col] == row ? 1 : 0);
    }
}