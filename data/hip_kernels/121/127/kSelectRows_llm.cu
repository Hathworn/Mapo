#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSelectRows(float* source, float* target, float* indices, int nRowIs, int nCols, int nSourceRows){
    __shared__ int sourceRowIndices[32];
    const int startTargetRowI = blockIdx.x * 32;
    const int tid = threadIdx.x;
    const int localNRowIs = min(32, nRowIs - startTargetRowI);

    // Load row indices
    if (tid < localNRowIs){
        int idx = int(indices[startTargetRowI + tid]);
        if (idx < 0) idx += nSourceRows;
        sourceRowIndices[tid] = (idx < 0 || idx >= nSourceRows) ? -1 : idx;
    }
    __syncthreads();

    // Copy rows
    for (int i = 0; i < localNRowIs; i++){
        const int targetRowI = startTargetRowI + i;
        const int sourceRowI = sourceRowIndices[i];
        if (sourceRowI != -1) {
            // Copy valid rows
            for (int colI = tid; colI < nCols; colI += 32) {
                target[targetRowI * nCols + colI] = source[sourceRowI * nCols + colI];
            }
        } else {
            // Handle invalid rows (optimized)
            for (int colI = tid; colI < nCols; colI += 32) {
                target[targetRowI * nCols + colI] = -1.0 / 0.0;  // Use NaN
            }
        }
    }
}