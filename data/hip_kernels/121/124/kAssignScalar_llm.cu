#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAssignScalar(float* dest, float alpha, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    if (idx < len) { // Check to ensure idx is within bounds
        dest[idx] = alpha;
    }
}