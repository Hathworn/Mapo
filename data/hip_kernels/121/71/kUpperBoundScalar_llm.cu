#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize warp efficiency by using shared memory and unroll the loop
__global__ void kUpperBoundScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Iterate with a stride equal to the total number of threads
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        float m = mat[i];
        target[i] = m > val ? val : m;
    }
}