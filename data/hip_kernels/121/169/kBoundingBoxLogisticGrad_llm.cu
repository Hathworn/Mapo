#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kBoundingBoxLogisticGrad( float* mat, int* bbox, int* label, int* seg, float* indices, float* width_offset, float* height_offset, int size, int width, int height, int depth, float scale_width, float scale_height, float* grad) {
    const int color = blockIdx.z;

    // Optimize thread and block calculation: Calculate indices efficiently
    const int image_id = threadIdx.x + blockIdx.x * blockDim.x;
    const int row = threadIdx.y + blockIdx.y * blockDim.y;

    int num_bboxes = 0, num_bboxes_of_this_depth = 0, num_bboxes_of_this_depth_inside = 0;
    if (image_id < size && row < height && color < depth) {
        int src_image_id = (int)indices[image_id];
        int src_col = (int)(scale_width * image_id);
        int src_row = (int)(scale_height * row);
        
        int start = seg[src_image_id];
        int end = seg[src_image_id + 1];
        int x1, y1, x2, y2, l, inside;
        for (int box_id = start; box_id < end; box_id++) {
            l = label[box_id];
            x1 = bbox[box_id << 2] - width_offset[image_id];
            y1 = bbox[(box_id << 2) + 1] - height_offset[image_id];
            x2 = bbox[(box_id << 2) + 2] - width_offset[image_id];
            y2 = bbox[(box_id << 2) + 3] - height_offset[image_id];
            inside = (src_col >= x1 && src_col <= x2 && src_row >= y1 && src_row <= y2) ? 1:0;
            num_bboxes += inside;
            num_bboxes_of_this_depth += (l == color) ? 1: 0;
            num_bboxes_of_this_depth_inside += (inside == 1 && l == color) ? 1: 0;
        }
    }
    
    // Calculate global index more efficiently
    unsigned long i = image_id + size * (row + height * color);
    __syncthreads();
    
    if (image_id < size && row < height && color < depth) {
        if (num_bboxes > 0) {
            grad[i] = (num_bboxes_of_this_depth_inside > 0) ? (mat[i] - 1) : 0;
        } else {
            grad[i] = (num_bboxes_of_this_depth > 0) ? mat[i] : 0;
        }
    }
}