#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAccumulateColumns(float* mat, float* indices, float* target, int mat_width, int target_width, int height, float mult, int avg) {
    const int row = gridDim.x * blockIdx.y + blockIdx.x;
    const int column = threadIdx.x;
    
    if (row < height && column < target_width) {
        float cur_sum = 0.0f;
        unsigned int count = 0;

        // Use shared memory to reduce global memory access
        extern __shared__ float shared_indices[];
        if (threadIdx.x < mat_width) {
            shared_indices[threadIdx.x] = indices[threadIdx.x];
        }
        __syncthreads();

        for (unsigned int i = 0; i < mat_width; i++) {
            float idx = shared_indices[i];
            float value = mat[row + i * height];
            if ((int)idx == column) {
                count++;
                cur_sum += value;
            }
        }

        // Use a ternary operator for concise division
        target[row + height * column] = mult * cur_sum / (avg ? max(count, 1) : 1);
    }
}