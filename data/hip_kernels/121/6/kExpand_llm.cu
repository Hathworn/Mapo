#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kExpand(float *images, float* targets, int num_images, int num_input_channels, int image_size_y, int image_size_x, int num_modules_y, int num_modules_x, int kernel_size_y, int kernel_size_x, int padding_y, int padding_x, int stride_y, int stride_x, int num_modules_batch, int module_id_offset) {
    // Calculate color and module indices
    int color = blockIdx.y;
    int src_module_id = module_id_offset + blockIdx.x;
    int dst_module_id = blockIdx.x;

    int module_id_x = src_module_id % num_modules_x;
    int module_id_y = src_module_id / num_modules_x;
    int startX = module_id_x * stride_x + padding_x;
    int startY = module_id_y * stride_y + padding_y;
    images += num_images * image_size_x * image_size_y * color;
    targets += num_images * (dst_module_id + num_modules_batch * (kernel_size_y * kernel_size_x * color));
    
    // Use shared memory for temporary storage within block (optional, for larger kernel sizes)
    // extern __shared__ float shared_targets[];

    for (int y = 0; y < kernel_size_y; y++) {
        int Y = startY + y;
        for (int x = 0; x < kernel_size_x; x++) {
            int X = startX + x;
            long target_id = num_images * num_modules_batch * (x + kernel_size_x * y);
            long source_id = num_images * (X + image_size_x * Y);
            
            // Use warp divergence to optimize boundary checks
            if (X < 0 || X >= image_size_x || Y < 0 || Y >= image_size_y) {
                for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
                    targets[target_id + im] = 0;
                }
            } else {
                for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
                    targets[target_id + im] = images[source_id + im];
                }
            }
        }
    }
    // Synchronize threads in the block (optional, if shared memory is used)
    // __syncthreads();
}