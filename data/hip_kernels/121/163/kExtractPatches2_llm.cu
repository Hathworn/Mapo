#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kExtractPatches2(float* images, float* patches, float* width_offset, float* height_offset, float* flip, int num_images, int img_width, int img_height, int patch_width, int patch_height, int num_colors) {
    int image_id = blockIdx.z % num_images;
    int color = blockIdx.z / num_images;
    int dest_col = blockIdx.x * blockDim.x + threadIdx.x;
    int dest_row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within patch boundaries
    if (dest_col < patch_width && dest_row < patch_height) {
        // Use local variables for offsets to optimize memory access
        int w_offset = (int)width_offset[image_id];
        int h_offset = (int)height_offset[image_id];
        
        // Calculate source positions
        int source_row = h_offset + dest_row;
        int source_col = (flip[image_id] > 0.5f) ? (img_width - (w_offset + dest_col) - 1) : (w_offset + dest_col);
        
        // Calculate indices and perform memory copy
        unsigned long dest_index = image_id + num_images * (dest_col + patch_width * (dest_row + patch_height * color));
        unsigned long source_index = source_col + img_width * (source_row + img_height * (color + num_colors * image_id));
        patches[dest_index] = images[source_index];
    }
}