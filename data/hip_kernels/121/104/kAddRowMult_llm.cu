#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddRowMult(float* mat, float* vec, float* tgtMat, float mult, unsigned int width, unsigned int height) {
    // Compute the overall index from block and thread indices
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    const unsigned int numElements = width * height;

    // Loop with a stride for better memory access pattern
    for (unsigned int i = idx; i < numElements; i += stride) {
        // Compute only if within bounds to avoid unnecessary operations
        const unsigned int rowIdx = i / height;
        tgtMat[i] = mat[i] + mult * vec[rowIdx];
    }
}