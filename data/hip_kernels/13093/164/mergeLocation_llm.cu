#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeLocation(const short2* __restrict__ loc_, float* __restrict__ x, float* __restrict__ y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;
    if (ptidx < npoints)
    {
        // Load loc value into a register for better performance
        short2 loc = loc_[ptidx];
        // Perform scaling and store results in output arrays
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}