#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate the global thread indices
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Boundary check to prevent out-of-bounds access
    if (i < h && j < w) 
    {
        // Compute the linear index
        const int pos = i * w + j;

        // Set value directly in global memory
        image[pos] = value;
    }
}