#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate i and j using built-in 2D block and thread indices
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit for out-of-bound indices, reducing warp divergence
    if (i < h && j < w) {
        const int pos = i * s + j;

        // Load scale once to reduce memory access
        float scale = normalization_factor[pos];
        
        // Perform normalization using pre-computed inverse scale
        float invScale = (scale != 0.0f) ? (1.0f / scale) : 1.0f;
        
        image[pos] *= invScale;
    }
}