#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // calculate target pixel position
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);

    int tx = (int)px, ty = (int)py;
    float value = src[image_row_offset + j];
    float weight;

    // optimize bounds checking by consolidating conditionals
    if (ty >= 0 && ty < h) {
        if (tx >= 0 && tx < w) {
            weight = dx * dy;
            _atomicAdd(dst + ty * image_stride + tx, value * weight);
            _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
        }
        if (tx - 1 >= 0) {
            weight = (1.0f - dx) * dy;
            _atomicAdd(dst + ty * image_stride + (tx - 1), value * weight);
            _atomicAdd(normalization_factor + ty * image_stride + (tx - 1), weight);
        }
    }
    if (ty - 1 >= 0 && ty - 1 < h) {
        if (tx - 1 >= 0) {
            weight = (1.0f - dx) * (1.0f - dy);
            _atomicAdd(dst + (ty - 1) * image_stride + (tx - 1), value * weight);
            _atomicAdd(normalization_factor + (ty - 1) * image_stride + (tx - 1), weight);
        }
        if (tx >= 0 && tx < w) {
            weight = dx * (1.0f - dy);
            _atomicAdd(dst + (ty - 1) * image_stride + tx, value * weight);
            _atomicAdd(normalization_factor + (ty - 1) * image_stride + tx, weight);
        }
    }
}