#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float softplus_kernel(float x, float threshold = 20) {
    // Optimize: Use condition ternary operator
    return (x > threshold) ? x : (x < -threshold) ? expf(x) : logf(expf(x) + 1);
}

__device__ float tanh_activate_kernel(float x) {
    // Optimize: Simplify expression
    return (2 / (1 + expf(-2 * x)) - 1);
}

__global__ void mish_kernel(const float *input, float *output, int num_elem) {
    // Calculate index
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    // Check bounds
    if (idx >= num_elem) return;
    // Compute Mish activation
    float sp = softplus_kernel(input[idx]);
    output[idx] = input[idx] * tanh_activate_kernel(sp);
}