#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_hardswish(const float *input_, float *output_, int n_data_size_)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if within bounds
    if (i < n_data_size_)
    {
        float val = input_[i];
        
        // Branchless computation using ternary operators
        output_[i] = (val >= 3.0f) ? val : (val <= -3.0f) ? 0.0f : val * (val + 3.0f) * 0.1666667f; // 1/6 is approximately 0.1666667
    }
}