#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_cmp_kernel(std::size_t n, int* aptr, int* bptr, int* rptr) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // Use shared memory for result
    __shared__ int block_result;
    if (threadIdx.x == 0) block_result = 0;
    __syncthreads();

    if (i < n && aptr[i] < bptr[i]) {
        atomicOr(&block_result, 1); // Use atomic operation for thread-safe update
    }
    __syncthreads();

    // Only one thread writes back the result to global memory
    if (threadIdx.x == 0 && block_result) {
        atomicOr(rptr, 1);
    }
}