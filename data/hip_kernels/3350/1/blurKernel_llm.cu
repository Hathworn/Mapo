#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define index(i, j, w)  ((i)*(w)) + (j)

__global__ void blurKernel(unsigned char *d_inputArray, unsigned char *d_outputArray, int w, int h, int blurSize) {
    // Calculate global thread coordinates
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread operates within image boundaries
    if (Col < w && Row < h) {
        int pixVal = 0;
        int pixels = 0;

        // Loop over the blur window
        for (int blurRow = -blurSize; blurRow <= blurSize; ++blurRow) {
            int curRow = Row + blurRow;
            if (curRow >= 0 && curRow < h) { // Valid row check outside inner loop
                for (int blurCol = -blurSize; blurCol <= blurSize; ++blurCol) {
                    int curCol = Col + blurCol;
                    // Validate column, calculate pixel value, increment pixel count
                    if (curCol >= 0 && curCol < w) {
                        pixVal += d_inputArray[index(curRow, curCol, w)];
                        pixels++;
                    }
                }
            }
        }

        // Write the averaged pixel value to the output array
        d_outputArray[index(Row, Col, w)] = (unsigned char)(pixVal / pixels);
    }
}