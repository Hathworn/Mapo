#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate thread's global index
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    // Initialize hiprand with unique seed for each thread
    hiprand_init(seed, tidx, 0, &state[tidx]);
}