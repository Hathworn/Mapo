#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define WARP_SIZE 32

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];
    
    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;
    
    // Use contiguous memory access patterns
    if (eidx < numElements) {
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            shmem[threadIdx.x] += a[eidx] * b[eidx];
        }
    }
    __syncthreads();
    
    // Reduce in shared memory using warp shuffle for better performance
    if (threadIdx.x < 256) shmem[threadIdx.x] += shmem[threadIdx.x + 256]; __syncthreads();
    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128]; __syncthreads();
    if (threadIdx.x < 64)  shmem[threadIdx.x] += shmem[threadIdx.x + 64];  __syncthreads();
    
    if (threadIdx.x < WARP_SIZE) {
        volatile float* smemWarp = shmem;
        int lane = threadIdx.x;
        if (lane < 32) {
            smemWarp[lane] += smemWarp[lane + 32];
            smemWarp[lane] += smemWarp[lane + 16];
            smemWarp[lane] += smemWarp[lane + 8];
            smemWarp[lane] += smemWarp[lane + 4];
            smemWarp[lane] += smemWarp[lane + 2];
            smemWarp[lane] += smemWarp[lane + 1];
        }
    }
    
    // Save result from the first thread only
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}