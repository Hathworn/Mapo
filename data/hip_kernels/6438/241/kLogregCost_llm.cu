#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // If the most probable label is not equal to the true label, then probability is zero
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            // Count number of labels whose probability is equal to the maximum
            int numMax = 0;
            int i = 0;
            // Unroll the loop to improve performance
            for (; i + 4 <= numOut; i += 4) {
                numMax += (probs[i * numCases + tx] == maxp);
                numMax += (probs[(i + 1) * numCases + tx] == maxp);
                numMax += (probs[(i + 2) * numCases + tx] == maxp);
                numMax += (probs[(i + 3) * numCases + tx] == maxp);
            }
            for (; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp);
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}