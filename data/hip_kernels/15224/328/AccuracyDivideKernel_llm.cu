#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Each thread updates a separate element in accuracy (assuming multiple elements)
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx == 0) {
        *accuracy /= N; // Ensure only one thread performs the division
    }
}