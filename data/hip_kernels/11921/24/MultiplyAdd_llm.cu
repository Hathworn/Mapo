#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Use blockDim.x and blockDim.y for more readability and flexibility
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate linear index
    int p = y * width + x;

    // Check if within bounds and compute
    if (x < width && y < height)
    {
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
    // Remove unnecessary __syncthreads() as there's no inter-thread dependency
}