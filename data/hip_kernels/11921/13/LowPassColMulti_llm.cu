#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LowPassColMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    __shared__ float data[CONVCOL_W*(CONVCOL_H + 2*RADIUS)];
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int block = blockIdx.x/(NUM_SCALES+3);
    const int scale = blockIdx.x - (NUM_SCALES+3)*block;
    const int miny = blockIdx.y*CONVCOL_H;
    const int maxy = min(miny + CONVCOL_H, height) - 1;
    const int totStart = miny - RADIUS;
    const int totEnd = maxy + RADIUS;
    const int colStart = block*CONVCOL_W + tx;
    const int colEnd = colStart + (height-1)*pitch;
    const int sStep = CONVCOL_W*CONVCOL_S;
    const int gStep = pitch*CONVCOL_S;
    float *kernel = d_Kernel + scale*16;
    const int size = pitch*height*scale;
    d_Result += size;
    d_Data += size;

    if (colStart < width) {
        float* sdata = data + ty*CONVCOL_W + tx;
        int gPos = colStart + (totStart + ty)*pitch;
        // Load data into shared memory with boundary handling
        for (int y = totStart + ty; y <= totEnd; y += blockDim.y) {
            if (y < 0)
                data[ty * CONVCOL_W + tx] = d_Data[colStart];
            else if (y >= height)
                data[ty * CONVCOL_W + tx] = d_Data[colEnd];
            else
                data[ty * CONVCOL_W + tx] = d_Data[gPos];
            sdata += sStep;
            gPos += gStep;
        }
    }
    __syncthreads();
    
    if (colStart < width) {
        float *sdata = data + ty*CONVCOL_W + tx;
        int gPos = colStart + (miny + ty)*pitch;
        // Perform convolution using pre-loaded data
        for (int y = miny + ty; y <= maxy; y += blockDim.y) {
            float sum = 0.0f;
            for (int k = 0; k < 5; ++k) {
                sum += kernel[k] * 
                      (sdata[(k * CONVCOL_W)] + 
                       sdata[((8-k) * CONVCOL_W)]); // symmetry utilization
            }
            sum += kernel[4] * sdata[4 * CONVCOL_W]; // middle element
            d_Result[gPos] = sum;
            sdata += sStep;
            gPos += gStep;
        }
    }
}