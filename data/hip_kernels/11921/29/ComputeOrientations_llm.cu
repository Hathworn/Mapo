#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32];
    __shared__ float gauss[16];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Initialize histogram and Gaussian values in shared memory
    hist[tx] = 0.0f;
    if (tx < 15)
        gauss[tx] = expf(-1.0f/(2.0f*3.0f*3.0f) * (tx - 7) * (tx - 7));
    __syncthreads();

    // Calculate pixel positions
    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;
    int px = xp & 15;
    int x = tx - px;

    // Load necessary data into shared memory
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = xp + x;
        int yi = yp + y;
        xi = max(0, min(xi, w - 1));
        yi = max(0, min(yi, h - 1));
        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    // Compute gradients and update histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = 16.0f * atan2f(dy, dx) / 3.1416f + 16.5f;
            bin = (bin == 32) ? 0 : bin;
            float grad = sqrtf(dx * dx + dy * dy);
            atomicAdd(&hist[bin], grad * gauss[x] * gauss[y]);
        }
    }
    __syncthreads();
    
    // Perform histogram smoothing
    float v = hist[tx];
    float val1 = hist[(tx + 1) & 31];
    float val2 = hist[(tx + 31) & 31];
    hist[tx] = 6 * v + 4 * (val1 + val2);
    __syncthreads();

    // Identify peaks and update orientations
    if (tx == 0) {
        float maxval1 = 0.0f, maxval2 = 0.0f;
        int i1 = -1, i2 = -1;
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }
        val1 = hist[(i1 + 1) & 31];
        val2 = hist[(i1 + 31) & 31];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Orient[bx] = 11.25f * ((peak < 0.0f) ? (peak + 32.0f) : peak);
        
        if (maxval2 >= 0.8f * maxval1) {
            val1 = hist[(i2 + 1) & 31];
            val2 = hist[(i2 + 31) & 31];
            peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Orient[bx + maxPts] = 11.25f * ((peak < 0.0f) ? (peak + 32.0f) : peak);
        } else {
            d_Orient[bx + maxPts] = -1;
        }
    }
}