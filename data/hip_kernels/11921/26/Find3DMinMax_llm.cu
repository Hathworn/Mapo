#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int height)
{
    // Reduced shared memory usage
    __shared__ float dataCache[9 * (MINMAX_SIZE + 2)];
    __shared__ float ymin[3 * (MINMAX_SIZE + 2)];
    __shared__ float ymax[3 * (MINMAX_SIZE + 2)];

    // Current tile and apron limits, relative to row start
    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_SIZE;
    const int xEndClamped = min(xStart + MINMAX_SIZE - 1, width - 1);
    int memWid = MINMAX_SIZE + 2;

    int memPos0 = (tx - WARP_SIZE + 1);
    int memPos1 = memPos0;
    int yq = 0;
    unsigned int output = 0;

    for (int y = 0; y < 34; y++) {
        output >>= 1;
        int memPos = yq * memWid + memPos0;
        int yp = 32 * blockIdx.y + y - 1;
        yp = clamp(yp, 0, height - 1);
        int readStart = yp * width;

        // Efficiently load data into shared memory
        if (tx >= (WARP_SIZE - 1)) {
            if (tx < MINMAX_SIZE + 2) {
                int xReadPos = xStart + tx - WARP_SIZE;
                if (xReadPos >= 0 && xReadPos < width) {
                    dataCache[memPos] = d_Data1[readStart + xReadPos];
                    dataCache[memPos + memWid] = d_Data2[readStart + xReadPos];
                    dataCache[memPos + 2 * memWid] = d_Data3[readStart + xReadPos];
                } else {
                    dataCache[memPos] = 0;
                    dataCache[memPos + memWid] = 0;
                    dataCache[memPos + 2 * memWid] = 0;
                }
            }
        }
        __syncthreads();

        int memPos2 = yq * memWid + tx;
        if (y > 1 && tx < MINMAX_SIZE + 2) {
            float min1 = fminf(fminf(dataCache[memPos0], dataCache[memPos1]), dataCache[memPos2]);
            float min2 = fminf(fminf(dataCache[memPos0 + memWid], dataCache[memPos1 + memWid]), dataCache[memPos2 + memWid]);
            float min3 = fminf(fminf(dataCache[memPos0 + 2 * memWid], dataCache[memPos1 + 2 * memWid]), dataCache[memPos2 + 2 * memWid]);
            float max1 = fmaxf(fmaxf(dataCache[memPos0], dataCache[memPos1]), dataCache[memPos2]);
            float max2 = fmaxf(fmaxf(dataCache[memPos0 + memWid], dataCache[memPos1 + memWid]), dataCache[memPos2 + memWid]);
            float max3 = fmaxf(fmaxf(dataCache[memPos0 + 2 * memWid], dataCache[memPos1 + 2 * memWid]), dataCache[memPos2 + 2 * memWid]);

            ymin[tx] = min1;
            ymin[tx + memWid] = fminf(fminf(min1, min2), min3);
            ymin[tx + 2 * memWid] = min3;
            ymax[tx] = max1;
            ymax[tx + memWid] = fmaxf(fmaxf(max1, max2), max3);
            ymax[tx + 2 * memWid] = max3;
        }
        __syncthreads();

        if (y > 1 && tx < MINMAX_SIZE) {
            int xWritePos = xStart + tx;
            if (xWritePos <= xEndClamped) {
                float minv = fminf(ymin[tx + memWid], ymin[tx + memWid + 2]);
                minv = fminf(minv, ymin[tx + 1]);
                minv = fminf(fminf(minv, ymin[tx + 1 + memWid]), dataCache[memPos0 + 1 + memWid]);
                minv = fminf(minv, dataCache[memPos2 + 1 + memWid]);
                minv = fminf(minv, d_ConstantA[1]);
                float maxv = fmaxf(ymax[tx + memWid], ymax[tx + memWid + 2]);
                maxv = fmaxf(maxv, ymax[tx + 1]);
                maxv = fmaxf(fmaxf(maxv, ymax[tx + 1 + memWid]), dataCache[memPos0 + 1 + memWid]);
                maxv = fmaxf(maxv, dataCache[memPos2 + 1 + memWid]);
                maxv = fmaxf(maxv, d_ConstantA[0]);
                if (dataCache[memPos1 + 1 + memWid] < minv || dataCache[memPos1 + 1 + memWid] > maxv)
                    output |= 0x80000000;
            }
        }
        __syncthreads();

        memPos0 = memPos1;
        memPos1 = memPos2;
        yq = (yq < 2 ? yq + 1 : 0);
    }
    if (tx < MINMAX_SIZE && xStart + tx < width) {
        int writeStart = blockIdx.y * width + xStart + tx;
        d_Result[writeStart] = output;
    }
}