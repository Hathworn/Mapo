#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Calculating the 3D thread and block indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;

    // Calculate flattened index for accessing arrays
    int sz = height * pitch;
    int p = z * sz + y * pitch + x;

    // Check boundaries before performing operations
    if (x < width && y < height)
    {
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
    // No need for __syncthreads() as there is no inter-thread dependency
}
```
