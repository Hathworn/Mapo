#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];

    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    int p = y * pitch + x;

    // Initialize shared memory with comparison for x < width
    float val = (x < width) ? d_Data[p] : d_Data[p - x];
    minvals[tx] = val;
    maxvals[tx] = val;

    // Use stride loop for memory access to reduce divergence
    for (int ty = 1; ty < 16; ty++) {
        p += pitch;
        if (x < width) {
            val = d_Data[p];
            minvals[tx] = fminf(minvals[tx], val);  // Use fminf for min values
            maxvals[tx] = fmaxf(maxvals[tx], val);  // Use fmaxf for max values
        }
    }

    __syncthreads(); // Synchronize threads before reduction

    // Use binary tree reduction pattern
    for (int d = 64; d > 0; d >>= 1) {
        if (tx < d) {
            minvals[tx] = fminf(minvals[tx], minvals[tx + d]);
            maxvals[tx] = fmaxf(maxvals[tx], maxvals[tx + d]);
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}