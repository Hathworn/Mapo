#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    // Calculate global thread's x and y index
    const int x = blockIdx.x * SUBTRACT_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACT_H + threadIdx.y;

    // Compute linear index
    int p = y * pitch + x;

    // Check if within bounds, perform subtraction if valid
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // Removed unnecessary __syncthreads();
}