#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;

    // Calculate row and column index based on block and thread to improve access pattern
    for (int global_y = blockIdx.y * (h / (32 * gridDim.y)); global_y < (blockIdx.y + 1) * (h / (32 * gridDim.y)); global_y++) {
        for (int x = blockIdx.x * 16; x < (blockIdx.x + 1) * 16 && x < w; x += 16) {
            unsigned int val = minmax[global_y * w + x + tx];

            // Process all bits in `val`
            for (int k = 0; k < 32 && numPts < maxPts; k++) {
                if (val & 0x1) {
                    ptrs[16 * numPts + tx] = (global_y * 32 + k) * w + x + tx;
                    numPts++;
                }
                val >>= 1;
            }
        }
    }
}