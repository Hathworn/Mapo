#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;
    // Use shared memory to reduce global memory access
    __shared__ unsigned int shared_val[32];
    
    for (int y = 0; y < h / 32; y++) {
        for (int x = 0; x < w; x += 16) {
            if (tx == 0) {
                shared_val[0] = minmax[y * w + x]; // Load value into shared memory
            }
            __syncthreads();
            unsigned int val = shared_val[0];

            if (val) {
                for (int k = 0; k < 32; k++) {
                    if ((val & 0x1) && (numPts < maxPts)) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
            __syncthreads(); // Ensure all threads are done before next iteration
        }
    }
}