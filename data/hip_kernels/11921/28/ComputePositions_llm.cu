#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Define block size
#define POSBLK_SIZE 128

__global__ void ComputePositions(float *g_Data1, float *g_Data2, float *g_Data3, int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
    int i = blockIdx.x * POSBLK_SIZE + threadIdx.x;
    if (i >= numPts)
        return;
    
    int p = d_Ptrs[i];
    
    // Preload common data to shared memory for coalesced access and reduce global memory accesses
    float val0 = g_Data2[p];
    float val1 = g_Data2[p-1];
    float val2 = g_Data2[p+1];
    float val3 = g_Data2[p-w];
    float val4 = g_Data2[p+w];
    float val5 = g_Data3[p];
    float val6 = g_Data1[p];
    
    // Compute gradient and hessian elements
    float dx = 0.5f * (val2 - val1);
    float dxx = 2.0f * val0 - val1 - val2;
    float dy = 0.5f * (val4 - val3);
    float dyy = 2.0f * val0 - val3 - val4;
    float ds = 0.5f * (val6 - val5);
    float dss = 2.0f * val0 - val5 - val6;
    
    // Compute mixed derivatives
    float dxy = 0.25f * (g_Data2[p+w+1] + g_Data2[p-w-1] - g_Data2[p-w+1] - g_Data2[p+w-1]);
    float dxs = 0.25f * (g_Data3[p+1] + g_Data1[p-1] - g_Data1[p+1] - g_Data3[p-1]);
    float dys = 0.25f * (g_Data3[p+w] + g_Data1[p-w] - g_Data3[p-w] - g_Data1[p+w]);
    
    // Compute determinant and inverse matrix components
    float idxx = dyy * dss - dys * dys;
    float idxy = dys * dxs - dxy * dss;
    float idxs = dxy * dys - dyy * dxs;
    float idyy = dxx * dss - dxs * dxs;
    float idys = dxy * dxs - dxx * dys;
    float idss = dxx * dyy - dxy * dxy;
    
    float det = idxx * dxx + idxy * dxy + idxs * dxs;
    float idet = 1.0f / det;
    
    // Compute position deltas using inverse Hessian
    float pdx = idet * (idxx * dx + idxy * dy + idxs * ds);
    float pdy = idet * (idxy * dx + idyy * dy + idys * ds);
    float pds = idet * (idxs * dx + idys * dy + idss * ds);
    
    // Check condition and fix deltas if outside threshold
    if (pdx < -0.5f || pdx > 0.5f || pdy < -0.5f || pdy > 0.5f || pds < -0.5f || pds > 0.5f)
    {
        pdx = __fdividef(dx, dxx);
        pdy = __fdividef(dy, dyy);
        pds = __fdividef(ds, dss);
    }
    
    float dval = 0.5f * (dx * pdx + dy * pdy + ds * pds);
    
    // Write results to output memory
    d_Sift[i + 0 * maxPts] = (p % w) + pdx;
    d_Sift[i + 1 * maxPts] = (p / w) + pdy;
    d_Sift[i + 2 * maxPts] = d_ConstantA[0] * exp2f(pds * d_ConstantB[0]);
    d_Sift[i + 3 * maxPts] = val0 + dval;
    
    float tra = dxx + dyy;
    det = dxx * dyy - dxy * dxy;
    
    // Check the contrast threshold
    d_Sift[i + 4 * maxPts] = __fdividef(tra * tra, det);
}