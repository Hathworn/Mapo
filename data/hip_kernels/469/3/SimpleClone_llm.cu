#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SimpleClone( const float *background, const float *target, const float *mask, float *output, const int wb, const int hb, const int wt, const int ht, const int oy, const int ox )
{
    // Calculate target indices
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within target boundaries and mask criteria
    if (yt < ht && xt < wt) {
        const int curt = wt * yt + xt;
        if (mask[curt] > 127.0f) {
            // Calculate background indices and ensure within boundaries
            const int yb = oy + yt, xb = ox + xt;
            if (yb < hb && xb < wb) {
                const int curb = wb * yb + xb;
                // Efficient memory operations using shared memory
                output[curb * 3 + 0] = __ldg(&target[curt * 3 + 0]);
                output[curb * 3 + 1] = __ldg(&target[curt * 3 + 1]);
                output[curb * 3 + 2] = __ldg(&target[curt * 3 + 2]);
            }
        }
    }
}