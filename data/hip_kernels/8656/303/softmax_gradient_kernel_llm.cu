#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    // Offsetting pointer to the current batch block
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;

    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    float sum = 0.0f;
    // Unrolling loop for better performance
    for (int i = idx; i < dim; i += blockDim.x) {
        sum += dY[i] * Y[i];
    }
    reduction_buffer[idx] = sum;
    __syncthreads();

    // Using warp-level reduction for faster processing
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        if (idx < offset)
            reduction_buffer[idx] += reduction_buffer[idx + offset];
        __syncthreads();
    }

    float tmp = reduction_buffer[0];
    // Compute gradient
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}