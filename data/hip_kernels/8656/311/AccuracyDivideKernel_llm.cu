#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use blockIdx.x and threadIdx.x to allow parallel execution
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        *accuracy /= N;
    }
}