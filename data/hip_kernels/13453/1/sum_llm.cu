#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(int *dest, int a, int b) {
    // Utilize block and thread indexing for scaling
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) { // Only the first thread computes the result
        *dest = a + b;
    }   
}