#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vector_add(double const *A_dev, double const *B_dev, double *C_dev, int const N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Ensure index is within bounds and perform vector addition
    if (i < N) {
        C_dev[i] = A_dev[i] + B_dev[i];
    }
}