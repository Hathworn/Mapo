#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float* __restrict__ gradInput_data, const float* __restrict__ gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x + (threadIdx.y + blockDim.y * blockIdx.y) * (blockDim.x * gridDim.x);
    if (ii >= no_elements) return;

    // Using shared memory to reduce global memory accesses
    __shared__ float cache[1024]; // Cache for accumulating results
    cache[threadIdx.y * blockDim.x + threadIdx.x] = 0.0f;

    for (int i = 0; i < scale_factor; i++) {
        for (int j = 0; j < scale_factor; j++) {
            int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
            cache[threadIdx.y * blockDim.x + threadIdx.x] += gradOutput_data[ipidx];
        }
    }

    // Synchronize threads to ensure all additions to cache are complete
    __syncthreads();

    // Write accumulated result to global memory
    gradInput_data[ii] = cache[threadIdx.y * blockDim.x + threadIdx.x];
}