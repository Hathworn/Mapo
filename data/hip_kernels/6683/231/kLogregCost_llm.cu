#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Use ternary operator for concise condition assignment
        correctProbs[tx] = (labelp != maxp)? 0 : 
        ({
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp);
            }
            1.0f / float(numMax);
        });
    }
}