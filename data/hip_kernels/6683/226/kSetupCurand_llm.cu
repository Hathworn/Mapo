#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel for setting up CURAND states
__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate unique global thread index
    const uint globalThreadIdx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Each thread gets a unique sequence number, no offset
    if (globalThreadIdx < MAX_NUM_THREADS) { // Ensure thread index is within bounds
        hiprand_init(seed, globalThreadIdx, 0, &state[globalThreadIdx]);
    }
}