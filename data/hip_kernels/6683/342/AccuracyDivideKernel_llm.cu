#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use block and thread id to ensure only the first thread performs the division
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N; // Single thread performs division
    }
}