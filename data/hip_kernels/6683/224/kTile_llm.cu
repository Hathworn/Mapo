#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Calculate corresponding coordinates in source
        const uint srcY = (i / tgtWidth) % srcHeight;
        const uint srcX = (i % tgtWidth) % srcWidth;
        // Perform tiling operation
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}