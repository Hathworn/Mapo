#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // Perform reduction within the block
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Only the first warp needs to perform reduction
    if (idx < 32) {
        #pragma unroll
        for (int i = 32; i < SOFTMAX_NUM_THREADS; i += 32) {
            tmp += reduction_buffer[idx + i];
        }
        reduction_buffer[idx] = tmp;

        #pragma unroll
        for (int offset = 16; offset > 0; offset /= 2) {
            tmp += __shfl_down_sync(0xFFFFFFFF, tmp, offset);
        }

        if (idx == 0) {
            reduction_buffer[0] = tmp;
        }
    }
    __syncthreads();

    // Compute gradient
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}