#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Perform division on the device, avoiding race conditions
    if (threadIdx.x == 0) {
        *accuracy /= static_cast<float>(N);
    }
}