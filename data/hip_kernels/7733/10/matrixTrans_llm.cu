#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixTrans(double *M, double *MT, int rows, int cols)
{
    // Use shared memory for coalesced accesses and to minimize global memory transactions
    __shared__ double tile[32][32 + 1]; // +1 to avoid bank conflicts

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < rows && col < cols) {
        // Load data into shared memory
        tile[threadIdx.y][threadIdx.x] = M[col + row * cols];
    }
    __syncthreads(); // Ensure all data is loaded before writing transposed

    row = blockIdx.y * blockDim.y + threadIdx.x;
    col = blockIdx.x * blockDim.x + threadIdx.y;

    if (row < cols && col < rows) {
        // Write data from shared memory to transposed matrix
        MT[col + row * rows] = tile[threadIdx.x][threadIdx.y];
    }
}