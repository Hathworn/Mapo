#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//Library Definition

//Constant Definition
#define PI 3.141592654
#define blocksize 32
#define Repetitions 8192

//Print matrix into standard output
void print(double * M,int cols,int rows);
void dot(double * a,double * b, double & c, int cols);
void Create_New_Matrix(double * M,double * New,int * vec, int p0, int pp,int nn);

/*
DEVICE FUNCTIONS
*/

//Matrix transposition (Rows and Cols of M)

__global__ void gaussjordan(double *A, double *I, int nn, int i)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Using shared memory optimization and removing redundant boundary checks
    __shared__ double sharedA[blocksize][blocksize];
    __shared__ double sharedI[blocksize][blocksize];

    if (x < nn && y < nn)
    {
        sharedA[threadIdx.x][threadIdx.y] = A[x*nn + y];
        sharedI[threadIdx.x][threadIdx.y] = I[x*nn + y];
    }

    __syncthreads();

    if (x < nn && y < nn && x != i)
    {
        double factor = sharedA[threadIdx.x][i];
        if (y == i)
        {
            sharedI[threadIdx.x][threadIdx.y] -= sharedI[i][threadIdx.y] * factor;
        }
        else
        {
            sharedA[threadIdx.x][threadIdx.y] -= sharedA[i][threadIdx.y] * factor;
            sharedI[threadIdx.x][threadIdx.y] -= sharedI[i][threadIdx.y] * factor;
        }

        // Write updated results back to global memory
        A[x*nn + y] = sharedA[threadIdx.x][threadIdx.y];
        I[x*nn + y] = sharedI[threadIdx.x][threadIdx.y];
    }
}