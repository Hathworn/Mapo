#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMul(double *a, double *b, double *C, int cols, int rows, int cols2) 
{
    // Calculate global row and column index using threadIdx.x and threadIdx.y
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // Check the boundary condition
    if (row < rows && col < cols) {
        double sum = 0;  // Initialize accumulator for the dot product
        // Use a loop to accumulate the dot product
        for (int k = 0; k < cols2; k++) {
            sum += b[k * cols + col] * a[row * cols2 + k];
        }
        // Write the computed value to the result matrix
        C[row * cols + col] = sum;
    }
}