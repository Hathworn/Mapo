#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nodiag_normalize(double *A, double *I, int nn, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check within matrix bounds and within same row, not diagonal
    if (x == i && x != y && y < nn) {
        double diagElement = A[i*nn + i]; // Cache diagonal element
        I[x*nn + y] /= diagElement;
        A[x*nn + y] /= diagElement;
    }
}