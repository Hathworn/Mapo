#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaussjordan(double *A, double *I, int nn, int i)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure threads operate within the matrix bounds
    if (x < nn && y < nn) {
        // Skip the row with index 'i'
        if (x != i) {
            // Perform the Gauss-Jordan elimination on matrix I
            I[x * nn + y] -= I[i * nn + y] * A[x * nn + i];
            // Perform the Gauss-Jordan elimination on matrix A, avoiding the pivot column
            if (y != i) {
                A[x * nn + y] -= A[i * nn + y] * A[x * nn + i];
            }
        }
    }
}