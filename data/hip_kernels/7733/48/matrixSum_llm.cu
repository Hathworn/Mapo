#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixSum(const double * M1, const double * M2, double * Msum, double alpha, double beta, int rows, int cols)
{
    // Calculate the row and column index using the optimal thread and block configuration
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // Proceed with computation only if the thread indices are within matrix bounds
    if (row < rows && col < cols) {
        // Use shared memory (if applicable) for faster data access; in this example, direct computation is used
        Msum[row + col * rows] = alpha * M1[row + col * rows] + beta * M2[row + col * rows];
    }
}