#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nodiag_normalize(double *A, double *I, int nn, int i) {
    // Calculate the thread's unique x-coordinate
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the thread's unique y-coordinate
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Skip unnecessary condition check, reducing redundant computation
    if (x < nn && y < nn && x == i && x != y) {
        double inverse = 1.0 / A[i * nn + i]; // Precompute inverse for reuse
        I[x * nn + y] *= inverse;             // Apply the inverse
        A[x * nn + y] *= inverse;             // Apply the inverse
    }
}