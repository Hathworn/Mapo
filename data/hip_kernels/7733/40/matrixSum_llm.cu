#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixSum(const double * M1, const double * M2, double * Msum, double alpha, double beta, int rows, int cols)
{
    // Utilize shared memory for faster access
    __shared__ double sharedM1[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ double sharedM2[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < rows && col < cols) {
        // Load data into shared memory
        sharedM1[threadIdx.x][threadIdx.y] = M1[row + col * rows];
        sharedM2[threadIdx.x][threadIdx.y] = M2[row + col * rows];

        __syncthreads(); // Ensure all threads have loaded their data

        // Compute using shared memory
        Msum[row + col * rows] = alpha * sharedM1[threadIdx.x][threadIdx.y] + beta * sharedM2[threadIdx.x][threadIdx.y];
    }
}
```
