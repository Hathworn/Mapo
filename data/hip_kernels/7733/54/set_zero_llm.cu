#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_zero(float *A, float *I, int n, int i) {
    // Optimize by combining condition checks to reduce branching
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < n && y < n && x != i && y == i) {
        A[x * n + y] = 0;
    }
}