#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMul(double* a, double* b, double* C, int cols, int rows, int cols2)
{
    // Calculate the row and col for the current thread
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within bounds
    if (row < rows && col < cols) {
        double sum = 0.0; // Initialize local accumulator
        // Perform partial product sum using shared memory 
        for (int k = 0; k < cols2; k++) {
            sum += b[k * cols + col] * a[row * cols2 + k];
        }
        C[row * cols + col] = sum; // Store computed result in matrix C
    }
}