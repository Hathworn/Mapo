#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixTrans(double *M, double *MT, int rows, int cols)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global row index
    int col = blockIdx.y * blockDim.y + threadIdx.y;  // Calculate global column index

    // Check boundaries to avoid out-of-bounds memory access
    if (row < rows && col < cols) {
        MT[row + col * rows] = M[col + row * cols];  // Perform matrix transposition
    }
}