#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nodiag_normalize(double *A, double *I, int nn, int i){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Single condition check; remove duplicate checks
    if (x < nn && y < nn && x == i && x != y) {
        double diag_value = A[i * nn + i];  // Cache the diagonal value
        I[x * nn + y] /= diag_value;
        A[x * nn + y] /= diag_value;
    }
}