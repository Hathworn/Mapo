#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMul(double *a, double *b, double *C, int cols, int rows, int cols2)
{
    // Use shared memory for faster access
    extern __shared__ double shared[];
    double* sharedA = shared;
    double* sharedB = shared + blockDim.x * blockDim.y;

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    double sum = 0.0;

    if (row < rows && col < cols) {
        for (int k = 0; k < cols2; k += blockDim.x) {
            if (threadIdx.x + k < cols2 && col < cols) {
                sharedA[threadIdx.y * blockDim.x + threadIdx.x] = a[row * cols2 + threadIdx.x + k];
                sharedB[threadIdx.y * blockDim.x + threadIdx.x] = b[(threadIdx.x + k) * cols + col];
            } else {
                sharedA[threadIdx.y * blockDim.x + threadIdx.x] = 0.0;
                sharedB[threadIdx.y * blockDim.x + threadIdx.x] = 0.0;
            }
            __syncthreads();

            for (int n = 0; n < blockDim.x; ++n) {
                sum += sharedA[threadIdx.y * blockDim.x + n] * sharedB[n * blockDim.x + threadIdx.x];
            }
            __syncthreads();
        }
        C[row * cols + col] = sum;
    }
}