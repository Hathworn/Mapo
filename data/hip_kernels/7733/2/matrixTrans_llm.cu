#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//Library Definition

//Constant Definition
#define PI 3.141592654
#define blocksize 32
#define Repetitions 8192

//Print matrix into standard output
void print(double * M, int cols, int rows);
void dot(double * a, double * b, double & c, int cols);
void Create_New_Matrix(double * M, double * New, int * vec, int p0, int pp, int nn);

/*
DEVICE FUNCTIONS
*/

//Matrix transposition (Rows and Cols of M)

__global__ void matrixTrans(double * M, double * MT, int rows, int cols)
{
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure that threads do not go out of bounds
    if (row < rows && col < cols) 
    {
        // Coalesced memory access for better performance
        int input_idx = col + row * cols;
        int output_idx = row + col * rows;
        MT[output_idx] = M[input_idx];
    }
}