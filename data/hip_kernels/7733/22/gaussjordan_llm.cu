#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaussjordan(double *A, double *I, int nn, int i)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < nn && y < nn) {
        // Perform row operations if x is not the pivot row i
        if (x != i) {
            I[x * nn + y] -= I[i * nn + y] * A[x * nn + i];
            // Perform column operations if y is not the pivot column i
            if (y != i) {
                A[x * nn + y] -= A[i * nn + y] * A[x * nn + i];
            }
        }
    }
}