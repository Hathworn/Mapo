#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaussjordan(float *A, float *I, int n, int i)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < n && y < n) {
        if (x != i) {
            float Axi = A[x*n + i];
            float Ixi_y = I[i*n + y] * Axi; // Pre-compute I[i*n+y] * A[x*n+i]
            I[x*n + y] -= Ixi_y;           // Deduct in-place to avoid recalculation
            if (y != i) {
                A[x*n + y] -= A[i*n + y] * Axi; // Deduct in-place to avoid recalculation
            }
        }
    }
}