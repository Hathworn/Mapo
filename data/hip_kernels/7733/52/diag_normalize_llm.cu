#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diag_normalize(float *A, float *I, int n, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Optimize by combining nested conditions and redundant computations
    if (x == y && x < n && x == i) {
        float diag_value = A[i * n + i];
        I[x * n + y] /= diag_value;
        A[x * n + y] /= diag_value;
    }
}