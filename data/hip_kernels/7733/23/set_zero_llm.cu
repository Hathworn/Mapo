#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_zero(double *A, double *I, int nn, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Optimize by reducing nested conditions into a single if statement
    if (x < nn && y < nn && x != i && y == i) {
        A[x * nn + y] = 0;
    }
}