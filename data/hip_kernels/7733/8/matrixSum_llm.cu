#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Constant Definition
#define PI 3.141592654
#define blocksize 32
#define Repetitions 8192

// Print matrix into standard output
void print(double * M, int cols, int rows);
void dot(double * a, double * b, double & c, int cols);
void Create_New_Matrix(double * M, double * New, int * vec, int p0, int pp, int nn);

/*
DEVICE FUNCTIONS
*/

// Matrix transposition (Rows and Cols of M)

__global__ void matrixSum(const double * M1, const double * M2, double * Msum, double alpha, double beta, int rows, int cols)
{
    // Use shared memory to improve memory access performance
    __shared__ double M1_tile[blocksize][blocksize];
    __shared__ double M2_tile[blocksize][blocksize];

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < rows && col < cols) {
        // Load data into shared memory
        M1_tile[threadIdx.y][threadIdx.x] = M1[row + col * rows];
        M2_tile[threadIdx.y][threadIdx.x] = M2[row + col * rows];
        __syncthreads();

        // Perform computation using shared memory
        Msum[row + col * rows] = alpha * M1_tile[threadIdx.y][threadIdx.x] + beta * M2_tile[threadIdx.y][threadIdx.x];
    }
}