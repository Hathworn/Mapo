#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diag_normalize(double *A, double *I, int nn, int i) {
    // Single calculation for block matching x == i
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < nn && x == i) {
        I[x * nn + x] /= A[i * nn + i];
        A[x * nn + x] /= A[i * nn + i];
    }
}