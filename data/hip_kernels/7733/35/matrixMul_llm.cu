#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMul(double *a, double *b, double *C, int cols, int rows, int cols2) {
    // Get unique row and column index for this thread
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Allocate shared memory for tiles of a and b
    __shared__ double tileA[TILE_DIM][TILE_DIM];
    __shared__ double tileB[TILE_DIM][TILE_DIM];
    
    double value = 0;
    
    // Iterate over tiles
    for (int t = 0; t < (cols2 + TILE_DIM - 1) / TILE_DIM; ++t) {
        // Load data into tiles
        if (row < rows && t * TILE_DIM + threadIdx.y < cols2) {
            tileA[threadIdx.x][threadIdx.y] = a[row * cols2 + t * TILE_DIM + threadIdx.y];
        } else {
            tileA[threadIdx.x][threadIdx.y] = 0.0;
        }
        
        if (col < cols && t * TILE_DIM + threadIdx.x < cols2) {
            tileB[threadIdx.x][threadIdx.y] = b[(t * TILE_DIM + threadIdx.x) * cols + col];
        } else {
            tileB[threadIdx.x][threadIdx.y] = 0.0;
        }
        
        // Synchronize threads to ensure tiles are fully loaded
        __syncthreads();
        
        // Perform multiplication within the tile
        for (int k = 0; k < TILE_DIM; ++k) {
            value += tileA[threadIdx.x][k] * tileB[k][threadIdx.y];
        }
        
        // Synchronize before loading next tile
        __syncthreads();
    }
    
    // Write the result to global memory
    if (row < rows && col < cols) {
        C[row * cols + col] = value;
    }
}