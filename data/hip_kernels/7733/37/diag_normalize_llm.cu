#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diag_normalize(double *A, double *I, int nn, int i){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check for valid indices
    if (x < nn && y < nn) {
        // Check for diagonal element and matching index
        if (x == y && x == i) {
            double diag_value = A[i * nn + i];
            // Normalize matrix elements
            I[x * nn + y] /= diag_value;
            A[x * nn + y] /= diag_value;
        }
    }
}