#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMul(double *a, double *b, double *C, int cols, int rows, int cols2) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Transpose block indices to utilize coalesced memory access
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Transpose block indices to utilize coalesced memory access

    if (row < rows && col < cols) {
        double sum = 0; // Use a local variable to accumulate the sum
        for (int k = 0; k < cols2; k++) {
            sum += b[k * cols + col] * a[row * cols2 + k];
        }
        C[row * cols + col] = sum; // Assign accumulated sum to C
    }
}