#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diag_normalize(double *A, double *I, int nn, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Utilize shared memory to reduce redundant global memory access.
    __shared__ double diag;

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        diag = A[i * nn + i];
    }
    __syncthreads();

    if (x < nn && y < nn) {
        if (x == y && x == i) {
            I[x * nn + y] /= diag;
            A[x * nn + y] /= diag;
        }
    }
}