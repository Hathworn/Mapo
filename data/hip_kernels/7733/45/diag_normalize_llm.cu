#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diag_normalize(double *A, double *I, int nn, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;   // Compute global thread x-index
    int y = blockIdx.y * blockDim.y + threadIdx.y;   // Compute global thread y-index

    // Guard conditions
    if (x < nn && y < nn && x == y && x == i) {
        double diag_elem = A[i * nn + i];            // Load diagonal element
        I[x * nn + y] /= diag_elem;                  // Normalize I matrix element
        A[x * nn + y] /= diag_elem;                  // Normalize A matrix element
    }
}