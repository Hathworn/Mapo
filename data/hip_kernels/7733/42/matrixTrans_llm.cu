#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixTrans(double * M, double * MT, int rows, int cols)
{
    // Use shared memory to improve memory access efficiency
    __shared__ double tile[32][32 + 1]; // Add padding to avoid bank conflicts

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        tile[threadIdx.y][threadIdx.x] = M[col + row * cols];
    }

    __syncthreads();

    // Write transposed output
    row = blockIdx.x * blockDim.x + threadIdx.y; // Transpose block offset
    col = blockIdx.y * blockDim.y + threadIdx.x;

    if (row < cols && col < rows) {
        MT[row + col * cols] = tile[threadIdx.x][threadIdx.y];
    }
}