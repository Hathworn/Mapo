#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_zero(double *A, double *I, int nn, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds just once for performance
    if (x < nn && y < nn && x != i && y == i) {
        // Directly set A[x*nn + y] to zero when conditions are met
        A[x * nn + y] = 0;
    }
}