#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//Library Definition

//Constant Definition
#define PI 3.141592654
#define blocksize 32
#define Repetitions 8192

//Print matrix into standard output
void print(double * M,int cols,int rows);
void dot(double * a,double * b, double & c, int cols);
void Create_New_Matrix(double * M,double * New,int * vec, int p0, int pp,int nn);

/*
DEVICE FUNCTIONS
*/

//Matrix transposition (Rows and Cols of M)

__global__ void diag_normalize(double *A, double *I, int nn, int i)
{
    // Calculate global thread id
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Simplify condition and avoid redundancy
    if (x == y && x == i && x < nn) 
    {
        double factor = A[i*nn + i]; // Cache the value to reduce memory access
        I[x*nn + y] /= factor;
        A[x*nn + y] /= factor;
    }
}