#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nodiag_normalize(double *A, double *I, int nn, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure indices are within bounds and focus on row 'i' only when x == i
    if (x == i && y < nn && x != y) { 
        double reciprocal = 1.0 / A[i * nn + i]; // Cache reciprocal to minimize division operations
        I[x * nn + y] *= reciprocal;
        A[x * nn + y] *= reciprocal;
    }
}