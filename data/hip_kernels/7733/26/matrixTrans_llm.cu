#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixTrans(double* __restrict__ M, double* __restrict__ MT, int rows, int cols)
{
    // Use shared memory for better data locality
    __shared__ double tile[32][32 + 1];  // Padding added to avoid bank conflicts

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure threads within bounds before loading data into shared memory
    if (row < rows && col < cols) {
        tile[threadIdx.y][threadIdx.x] = M[col + row * cols];
    }
    __syncthreads();

    // Transpose using shared memory
    row = blockIdx.y * blockDim.y + threadIdx.x;
    col = blockIdx.x * blockDim.x + threadIdx.y;

    // Ensure threads are within bounds before writing data back to global memory
    if (row < cols && col < rows) {
        MT[row + col * cols] = tile[threadIdx.x][threadIdx.y];
    }
}