#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nodiag_normalize(double *A, double *I, int nn, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure x and y are within bounds, and only process off-diagonal elements of row i
    if (x < nn && y < nn && x == i && x != y) {
        double diag_value = A[i * nn + i];
        I[x * nn + y] /= diag_value;  // Normalize I matrix
        A[x * nn + y] /= diag_value;  // Normalize A matrix
    }
}