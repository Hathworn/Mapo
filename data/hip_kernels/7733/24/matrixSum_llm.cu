#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixSum(const double * M1, const double * M2, double * Msum, double alpha, double beta, int rows, int cols)
{
    // Calculate flattened thread ID for faster memory access
    int index = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * rows;
    
    // Check if index is within bounds
    if (index < rows * cols) {
        // Perform computation
        Msum[index] = alpha * M1[index] + beta * M2[index];
    }
}