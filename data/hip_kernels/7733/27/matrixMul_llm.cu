#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMul(double *a, double *b, double *C, int cols, int rows, int cols2) {
    // Calculate global row and column index for the element
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure the thread is within matrix bounds
    if (row < rows && col < cols) {
        double sum = 0.0; // Accumulate result in a local variable
        for (int k = 0; k < cols2; ++k) {
            sum += b[k * cols + col] * a[row * cols2 + k];
        }
        C[row * cols + col] = sum; // Assign accumulated sum to matrix C
    }
}