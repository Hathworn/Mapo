#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixSum(const double *M1, const double *M2, double *Msum, double alpha, double beta, int rows, int cols)
{
    // Use 1D grid and thread index for better memory coalescing
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate total elements in the matrix
    int totalElements = rows * cols;

    // Ensure the thread index is within bounds
    if (tid < totalElements) {
        Msum[tid] = alpha * M1[tid] + beta * M2[tid];
    }
}