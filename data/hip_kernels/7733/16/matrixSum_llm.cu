#include "hip/hip_runtime.h"
```c++
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixSum(const double * M1,const double * M2,double * Msum,double alpha,double beta, int rows, int cols)
{
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < rows && col < cols) {
        int index = row + col * rows; // Combine index calculations
        Msum[index] = alpha * M1[index] + beta * M2[index]; // Reuse index variable
    }
}