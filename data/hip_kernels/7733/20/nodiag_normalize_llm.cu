#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nodiag_normalize(double *A, double *I, int nn, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Perform index boundary check once and remove redundant checks
    if (x < nn && y < nn && x == i && x != y) {
        double factor = A[i * nn + i];  // Cache A[i*nn+i] to reduce global memory access
        I[x * nn + y] /= factor;
        A[x * nn + y] /= factor;
    }
}