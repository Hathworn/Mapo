#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_zero(double *A, double *I, int nn, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Use a single if condition to minimize branch divergence
    if (x < nn && y == i && x != i) {
        A[x * nn + y] = 0;
    }
}