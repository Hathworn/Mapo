#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_zero(double *A, double *I, int nn, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Use single condition to reduce branching
    if (x < nn && y == i && x != i) {
        A[x * nn + y] = 0;
    }
}