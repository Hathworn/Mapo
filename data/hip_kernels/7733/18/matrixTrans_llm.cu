#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixTrans(double * M, double * MT, int rows, int cols)
{
    // Use shared memory for faster access
    __shared__ double tile[32][32+1]; // Avoid bank conflicts with padding

    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIndex < cols && yIndex < rows) {
        int index_in = yIndex * cols + xIndex;
        tile[threadIdx.y][threadIdx.x] = M[index_in];
    }

    __syncthreads(); // Synchronize threads within block

    xIndex = blockIdx.y * blockDim.y + threadIdx.x; // Transpose block offset
    yIndex = blockIdx.x * blockDim.x + threadIdx.y;

    if (xIndex < rows && yIndex < cols) {
        int index_out = yIndex * rows + xIndex;
        MT[index_out] = tile[threadIdx.x][threadIdx.y];
    }
}