#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//Library Definition

//Constant Definition
#define PI 3.141592654
#define blocksize 32
#define Repetitions 8192

//Print matrix into standard output
void print(double * M, int cols, int rows);
void dot(double * a, double * b, double & c, int cols);
void Create_New_Matrix(double * M, double * New, int * vec, int p0, int pp, int nn);

/*
DEVICE FUNCTIONS
*/

//Matrix transposition (Rows and Cols of M)
__global__ void nodiag_normalize(double *A, double *I, int nn, int i)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure threads are within matrix dimensions and x is not equal to y
    if (x < nn && y < nn && x == i && x != y)
    {
        double diagValue = A[i * nn + i];
        
        // Perform normalization for the current element
        I[x * nn + y] /= diagValue;
        A[x * nn + y] /= diagValue;
    }
}