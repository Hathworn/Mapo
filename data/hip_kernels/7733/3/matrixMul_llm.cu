#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// Library Definition

// Constant Definition
#define PI 3.141592654
#define blocksize 32
#define Repetitions 8192

// Print matrix into standard output
void print(double * M, int cols, int rows);
void dot(double * a, double * b, double & c, int cols);
void Create_New_Matrix(double * M, double * New, int * vec, int p0, int pp, int nn);

/*
DEVICE FUNCTIONS
*/

// Optimized matrix multiplication kernel
__global__ void matrixMul(double * a, double * b, double * C, int cols, int rows, int cols2)
{
    // Using shared memory for efficient data reuse
    __shared__ double shared_a[blocksize][blocksize];
    __shared__ double shared_b[blocksize][blocksize];

    // Calculate row and column index
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    double sum = 0.0;

    // Loop over the tiles of the matrix
    for (int tileIdx = 0; tileIdx < (cols2 + blocksize - 1) / blocksize; ++tileIdx) {
        // Load elements into shared memory
        if (tileIdx * blocksize + threadIdx.y < cols2 && row < rows)
            shared_a[threadIdx.y][threadIdx.x] = a[row * cols2 + tileIdx * blocksize + threadIdx.x];
        else
            shared_a[threadIdx.y][threadIdx.x] = 0.0;

        if (tileIdx * blocksize + threadIdx.x < cols && col < cols)
            shared_b[threadIdx.y][threadIdx.x] = b[(tileIdx * blocksize + threadIdx.y) * cols + col];
        else
            shared_b[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        // Multiply the matrices
        for (int k = 0; k < blocksize; ++k) {
            sum += shared_a[threadIdx.y][k] * shared_b[k][threadIdx.x];
        }
        __syncthreads();  // Ensure memory coherence
    }

    // Write the result to the output matrix
    if (row < rows && col < cols) {
        C[row * cols + col] = sum;
    }
}