#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaussjordan(double *A, double *I, int nn, int i)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure indices are within bounds and skip unnecessary checks
    if (x < nn && y < nn && x != i) {
        double AI = A[x * nn + i];
        I[x * nn + y] -= I[i * nn + y] * AI;

        // Perform subtraction only when necessary
        if (y != i) {
            A[x * nn + y] -= A[i * nn + y] * AI;
        }
    }
}