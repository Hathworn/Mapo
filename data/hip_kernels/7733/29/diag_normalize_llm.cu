#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diag_normalize(double *A, double *I, int nn, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Combine conditions to reduce branching
    if (x == y && x < nn && x == i) {
        double diag_value = A[i * nn + i]; // Cache diagonal value
        I[x * nn + y] /= diag_value;
        A[x * nn + y] /= diag_value;
    }
}