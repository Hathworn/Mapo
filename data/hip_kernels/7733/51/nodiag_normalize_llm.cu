#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nodiag_normalize(float *A, float *I, int n, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure x and y are within bounds and x equals i, but x is not equal to y
    if (x == i && x != y && y < n) {
        float diag = A[i * n + i];  // Cache the diagonal element for reuse
        I[x * n + y] /= diag;
        A[x * n + y] /= diag;
    }
}