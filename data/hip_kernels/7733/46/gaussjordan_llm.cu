#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaussjordan(double *A, double *I, int nn, int i)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < nn && y < nn) {
        if (x != i) {
            I[x * nn + y] -= I[i * nn + y] * A[x * nn + i];  // Update matrix I
            if (y != i) {
                A[x * nn + y] -= A[i * nn + y] * A[x * nn + i];  // Update matrix A
            }
        }
    }
}