#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixTrans(float * M, float * MT)
{
    // Use shared memory to reduce global memory accesses
    __shared__ float tile[TILE_DIM][TILE_DIM+1];

    int x = blockIdx.x * TILE_DIM + threadIdx.x; // global row index
    int y = blockIdx.y * TILE_DIM + threadIdx.y; // global column index

    if (x < N && y < N)
    {
        // Load data from global to shared memory
        tile[threadIdx.y][threadIdx.x] = M[y * N + x];
    }
    
    __syncthreads();

    // Transpose within shared memory
    x = blockIdx.y * TILE_DIM + threadIdx.x; // transposed global row index
    y = blockIdx.x * TILE_DIM + threadIdx.y; // transposed global column index

    if (x < N && y < N)
    {
        // Store back to global memory from shared memory
        MT[y * N + x] = tile[threadIdx.x][threadIdx.y];
    }
}
```

Make sure to define `TILE_DIM` as a macro or constant value, which should ideally match the size of the blocks for optimal performance.