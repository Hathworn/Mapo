#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diag_normalize(double *A, double *I, int nn, int i) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check boundaries once and avoid redundant calculations
    if (x < nn && y < nn && x == y && x == i) {
        double diag_val = A[i * nn + i];
        I[x * nn + y] /= diag_val;
        A[x * nn + y] /= diag_val;
    }
}