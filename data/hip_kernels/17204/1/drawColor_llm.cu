#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*************************************************************************/
__global__ void drawColor(unsigned char* optr, const float* red, const float* green, const float* blue) {
    // Calculate unique thread index for flattened 2D grid
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // Use __saturatef to clamp values between 0 and 1
    float theRed = __saturatef(red[offset]);
    float theGreen = __saturatef(green[offset]);
    float theBlue = __saturatef(blue[offset]);

    // Use optimized packing to store results
    optr[offset * 4 + 0] = __float2uint_rd(theRed * 255.0f);
    optr[offset * 4 + 1] = __float2uint_rd(theGreen * 255.0f);
    optr[offset * 4 + 2] = __float2uint_rd(theBlue * 255.0f);
    optr[offset * 4 + 3] = 255; // Alpha (constant)
}