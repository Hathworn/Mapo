#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate unique thread index
    if (tid < N) {  // Ensure the index is within the range of the arrays
        c[tid] = a[tid] + b[tid];
    }
}