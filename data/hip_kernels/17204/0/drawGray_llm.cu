#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*****************************************************************************/

/*****************************************************************************/

/*****************************************************************************/
__global__ void drawGray(unsigned char* optr, const float* outSrc) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    if (x < gridDim.x * blockDim.x && y < gridDim.y * blockDim.y) {
        // Scale and clamp the value to [0, 1]
        float val = fminf(fmaxf((outSrc[offset] / 50.0f) + 0.5f, 0.0f), 1.0f);

        unsigned char pixelValue = static_cast<unsigned char>(255 * val);

        optr[offset * 4 + 0] = pixelValue; // red
        optr[offset * 4 + 1] = pixelValue; // green
        optr[offset * 4 + 2] = pixelValue; // blue
        optr[offset * 4 + 3] = 255;        // alpha (opacity)
    }
}