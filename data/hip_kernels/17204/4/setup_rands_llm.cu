#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_rands(hiprandState* rand, unsigned long seed, unsigned long N)
{
    // Calculate global thread index more efficiently
    unsigned long tid = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    // Initialize random state only if tid is within bounds
    if (tid < N) {
        hiprand_init(seed, tid, 0, &rand[tid]);
    }
}