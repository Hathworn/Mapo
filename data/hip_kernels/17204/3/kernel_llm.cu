#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float* red, float* green, float* blue, unsigned long N){

    // Calculate global thread ID in one step
    unsigned long tid = blockIdx.y * blockDim.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;

    if(tid < N){
        red[tid] = .5f;  // Use 'f' suffix for float literals
        blue[tid] = .7f;
        green[tid] = .2f;
    }
}