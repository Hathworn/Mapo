#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned char value(float n1, float n2, int hue) {
    if (hue > 360) hue -= 360;
    else if (hue < 0) hue += 360;

    if (hue < 60)
        return (unsigned char)(255 * (n1 + (n2 - n1) * hue / 60));
    if (hue < 180)
        return (unsigned char)(255 * n2);
    if (hue < 240)
        return (unsigned char)(255 * (n1 + (n2 - n1) * (240 - hue) / 60));
    return (unsigned char)(255 * n1);
}

__global__ void randomize(float* array, hiprandState* rand, unsigned long N) {
    // Calculate global thread ID using combined dimensions for reduced divergence
    unsigned long tid = blockIdx.x * blockDim.x * blockDim.y
                        + threadIdx.y * blockDim.x + threadIdx.x;

    if (tid < N) {
        hiprandState localState = rand[tid]; // Get local hiprandState as seed
        float theRand = hiprand_uniform(&localState); // Get a random value
        rand[tid] = localState; // Save the updated state

        array[tid] = theRand; // Assign the random value to the array
    }
}