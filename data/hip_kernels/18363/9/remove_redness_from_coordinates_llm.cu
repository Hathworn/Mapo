#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void remove_redness_from_coordinates( const unsigned int*  d_coordinates, 
                                                unsigned char* d_r, 
                                                unsigned char* d_b, 
                                                unsigned char* d_g, 
                                                unsigned char* d_r_output, 
                                                int num_coordinates, 
                                                int num_pixels_y, 
                                                int num_pixels_x, 
                                                int template_half_height, 
                                                int template_half_width )
{
    int nx = num_pixels_x;
    int ny = num_pixels_y;
    int imgSize = nx * ny;
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_index_1d < num_coordinates)
    {
        unsigned int image_index_1d = d_coordinates[imgSize - global_index_1d - 1];
        ushort2 image_index_2d = make_ushort2(image_index_1d % nx, image_index_1d / nx);

        // Pre-compute min/max boundaries for the template to reduce redundant calculations
        int min_y = max(0, image_index_2d.y - template_half_height);
        int max_y = min(ny - 1, image_index_2d.y + template_half_height);
        int min_x = max(0, image_index_2d.x - template_half_width);
        int max_x = min(nx - 1, image_index_2d.x + template_half_width);

        for (int y = min_y; y <= max_y; y++)
        {
            for (int x = min_x; x <= max_x; x++)
            {
                int image_offset_index_1d_clamped = y * nx + x;

                unsigned char g_value = d_g[image_offset_index_1d_clamped];
                unsigned char b_value = d_b[image_offset_index_1d_clamped];

                // Use bitwise shift for average calculation for integer division
                unsigned int gb_average = (g_value + b_value) >> 1;

                d_r_output[image_offset_index_1d_clamped] = (unsigned char)gb_average;
            }
        }
    }
}