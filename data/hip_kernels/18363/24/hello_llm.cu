#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Optimize I/O by minimizing printf usage in the kernel
    int tid = threadIdx.x; // Cache thread index to register
    if (tid < 32) {        // Limit output to the first 32 threads for efficiency
        printf("Hello world! I'm thread %d\n", tid);
    }
}