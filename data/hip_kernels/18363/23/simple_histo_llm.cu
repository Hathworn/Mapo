#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;

    // Ensure threadId does not exceed input size
    if (myId < BIN_COUNT) {
        int myItem = d_in[myId];
        int myBin = myItem % BIN_COUNT;

        // Use shared memory for bins to reduce global memory access
        extern __shared__ int shared_bins[];

        atomicAdd(&(shared_bins[myBin]), 1);
        __syncthreads();

        // Only one block reduces shared results to global memory
        atomicAdd(&(d_bins[myBin]), shared_bins[myBin]);
    }
}