#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth(float * v_new, const float * v) {
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x; // Correct index calculation
    int numThreads = blockDim.x * gridDim.x;
    
    if (myIdx >= numThreads) return; // Ensure index within bounds

    // Use shared memory for neighboring elements access if needed 
    extern __shared__ float shmem[]; 
    int localIdx = threadIdx.x;
    shmem[localIdx] = v[myIdx];
    
    __syncthreads(); // Synchronize threads in a block

    float myLeftElt = (localIdx == 0) ? v[myIdx - 1] : shmem[localIdx - 1];
    float myRightElt = (localIdx == blockDim.x - 1) ? v[myIdx + 1] : shmem[localIdx + 1];
    
    v_new[myIdx] = 0.25f * myLeftElt + 0.5f * shmem[localIdx] + 0.25f * myRightElt;
}