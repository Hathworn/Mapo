#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
    // Utilize warp shuffle for reduction
    p += __shfl_down_sync(0xFFFFFFFF, p, 16);
    p += __shfl_down_sync(0xFFFFFFFF, p, 8);
    p += __shfl_down_sync(0xFFFFFFFF, p, 4);
    p += __shfl_down_sync(0xFFFFFFFF, p, 2);
    p += __shfl_down_sync(0xFFFFFFFF, p, 1);
    
    if (threadIdx.x % 32 == 0) s[threadIdx.x / 32] = p;  // Store warp results to shared memory
    __syncthreads();

    // Reduce warp results stored in shared memory
    if (threadIdx.x < warpSize) {
        unsigned int val = (threadIdx.x < (blockDim.x / warpSize)) ? s[threadIdx.x] : 0;
        if (threadIdx.x < 16) {
            val += __shfl_down_sync(0xFFFFFFFF, val, 8);
            val += __shfl_down_sync(0xFFFFFFFF, val, 4);
            val += __shfl_down_sync(0xFFFFFFFF, val, 2);
            val += __shfl_down_sync(0xFFFFFFFF, val, 1);
        }
        if (threadIdx.x == 0) s[0] = val;  // Final result to s[0]
    }
    __syncthreads();

    return s[0];
}

__global__ void reduce(unsigned int * d_out_shared, const unsigned int * d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    if (t == 0) {
        *d_out_shared = sr;
    }
}