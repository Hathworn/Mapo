#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
{
    const float normalization_constant = 1.f / d_input_cdf[n - 1];
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce global memory access for normalization constant
    __shared__ float shared_normalization_constant;
    if (threadIdx.x == 0) {
        shared_normalization_constant = normalization_constant;
    }
    __syncthreads();

    if (global_index_1d < n)
    {
        float output_value = d_input_cdf[global_index_1d] * shared_normalization_constant;
        d_output_cdf[global_index_1d] = output_value;
    }
}