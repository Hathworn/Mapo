#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// http://en.wikipedia.org/wiki/Bitonic_sort

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    // Perform bitonic sort using parallel threads
    for (int stage = 0; stage <= 5; stage++)
    {
        int seq_len = 1 << (stage + 1);
        for (int substage = stage; substage >= 0; substage--)
        {
            int partner_distance = 1 << substage;
            int pair_index = tid ^ partner_distance;
        
            if (pair_index > tid)
            {
                bool ascending = ((tid & seq_len) == 0);
                // Swap elements by comparing
                if ((sdata[tid] > sdata[pair_index]) == ascending)
                {
                    float temp = sdata[tid];
                    sdata[tid] = sdata[pair_index];
                    sdata[pair_index] = temp;
                }
            }
            __syncthreads();  // Ensure all comparisons and swaps in this stage are done
        }
    }

    d_out[tid] = sdata[tid];
}