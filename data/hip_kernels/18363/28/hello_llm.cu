#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Improved performance by eliminating repeated blockIdx.x access
    int block_id = blockIdx.x;
    printf("Hello world! I'm a thread in block %d\n", block_id);
}