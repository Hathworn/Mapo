#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tonemap( float* d_x, float* d_y, float* d_log_Y, float* d_cdf_norm, float* d_r_new, float* d_g_new, float* d_b_new, float  min_log_Y, float  max_log_Y, float  log_Y_range, int    num_bins, int    num_pixels_y, int    num_pixels_x )
{
    int2 image_index_2d = make_int2( ( blockIdx.x * blockDim.x ) + threadIdx.x, ( blockIdx.y * blockDim.y ) + threadIdx.y );
    int  image_index_1d = ( num_pixels_x * image_index_2d.y ) + image_index_2d.x;

    // Ensure index is within bounds
    if ( image_index_2d.x < num_pixels_x && image_index_2d.y < num_pixels_y )
    {
        float x         = d_x[ image_index_1d ];
        float y         = d_y[ image_index_1d ];
        float log_Y     = d_log_Y[ image_index_1d ];
        
        // Calculate bin index efficiently
        int   bin_index = min( num_bins - 1, int(fmaf(num_bins, ( log_Y - min_log_Y ) / log_Y_range, 0.0f)));
        float Y_new     = d_cdf_norm[ bin_index ];

        // Efficiently compute new color values
        float X_new = x * ( Y_new / y );
        float Z_new = fmaf(-x - y, Y_new / y, Y_new / y);

        float r_new = fmaf(X_new, 3.2406f, fmaf(Y_new, -1.5372f, Z_new * -0.4986f));
        float g_new = fmaf(X_new, -0.9689f, fmaf(Y_new, 1.8758f, Z_new * 0.0415f));
        float b_new = fmaf(X_new, 0.0557f, fmaf(Y_new, -0.2040f, Z_new * 1.0570f));

        d_r_new[ image_index_1d ] = r_new;
        d_g_new[ image_index_1d ] = g_new;
        d_b_new[ image_index_1d ] = b_new;
    }
}