#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel to compute matrix offset using shared memory for efficiency
__global__ void ComputeOffsetOfMatrixAB(const int32_t* __restrict__ row_sum, const int32_t* __restrict__ col_sum, int32_t* __restrict__ output, int32_t K_A_B, int32_t N) {
    // Shared memory for row_sum to reduce global memory reads
    __shared__ int32_t shared_row_sum;
    
    if (threadIdx.x == 0) {
        shared_row_sum = row_sum[blockIdx.x];
    }
    __syncthreads();

    for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
        output[blockIdx.x * N + i] = K_A_B - shared_row_sum - col_sum[i];
    }
}