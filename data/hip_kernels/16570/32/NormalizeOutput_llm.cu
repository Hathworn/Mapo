#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Iterate over stride instead of recalculating each time
    for (; idx < num_elements; idx += stride) {
        int offset = idx * 3; // Calculate offset once per iteration
        to_normalize[offset] = batch_index;
        to_normalize[offset + 1] = class_index;
        to_normalize[offset + 2] = static_cast<int64_t>(original[idx]);
    }
}