#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    // Calculate global index for accessing arrays
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Iterate over multiple elements in case index < N
    for (int32_t i = index; i < N; i += gridDim.x * blockDim.x) {
        *(output + blockIdx.x * N + i) = -col_sum[i];
    }
}