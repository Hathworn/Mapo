#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Calculate the global thread index
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if the index is within bounds
    if (idx < N) {
        output[blockIdx.x * N + idx] = -row_sum[blockIdx.x];
    }
}