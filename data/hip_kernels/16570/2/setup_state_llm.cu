#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Utilize blockIdx and threadIdx to separate states and increase concurrency
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}