#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void colDiv(float* a, float* b, float* c, int M, int N) {
    int i = blockIdx.y * M + blockIdx.x * blockDim.x + threadIdx.x;
    // Avoid out-of-bound access
    if (i < M * N) {
        int idx_a = blockIdx.y * M + (i % M);
        int idx_b = i % M;
        c[i] = a[idx_a] / b[idx_b];
    }
}