#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecMult(float* a, float* b, float* c, const int N) {
    // Optimized index computation
    const int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    if (i < N) {
        c[i] = a[i] * b[i];
    }
}