#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce0(float *g_idata, float *g_odata, int N) {
    extern __shared__ float sdata[];

    // Each thread loads one element from global to shared memory
    int tid = threadIdx.x;
    int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    float mySum = (i < N) ? g_idata[i] : 0.0f;
    if ((i + blockDim.x) < N) mySum += g_idata[i + blockDim.x];
    sdata[tid] = mySum;
    __syncthreads();

    // Optimize reduction using warp-level operations
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid < 32) {
        // Unrolling the last warp
        mySum = sdata[tid] + sdata[tid + 32];
        mySum += sdata[tid + 16];
        mySum += sdata[tid + 8];
        mySum += sdata[tid + 4];
        mySum += sdata[tid + 2];
        mySum += sdata[tid + 1];
        sdata[tid] = mySum;
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}