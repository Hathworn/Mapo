#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void colMul(float* a, float* b, float* c, int M, int N) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y;
    
    // Process only valid columns
    if (col < M) {
        int ind = col + row * M;
        c[ind] = a[ind] * b[col];
    }
}