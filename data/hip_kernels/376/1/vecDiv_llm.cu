#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecDiv(float* a, float* b, float* c, const int N) {
    // Optimize by calculating the global thread index in 3D for better utilization
    const int i = blockIdx.z * (gridDim.y * gridDim.x * blockDim.y * blockDim.x) 
                + blockIdx.y * (gridDim.x * blockDim.x * blockDim.y)
                + blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        // Use intrinsic for performance improvement
        c[i] = __fdividef(a[i], b[i]); // faster, less-accurate divide
    }
}