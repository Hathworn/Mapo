#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecEps(float* a, const int N) {
    // Calculate the global index
    const int i = blockIdx.y * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds before processing
    if (i < N) {
        // Check value with EPS and update if necessary
        if (a[i] < EPS) {
            a[i] = EPS;
        }
    }
}