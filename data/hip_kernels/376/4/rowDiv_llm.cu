#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rowDiv(float* a, float* b, float* c, int M, int N){
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread is within bounds
    if (i < M * N) {
        // Efficiently perform row-wise division
        c[i] = a[i] / b[blockIdx.x];
    }
}