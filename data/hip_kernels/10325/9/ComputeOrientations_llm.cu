#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOrientations(float *g_Data, float *d_Sift, int maxPts, int w, int h)
{
    __shared__ float data[16*15]; // Shared memory for data
    __shared__ float hist[32*13]; // Shared memory for histogram
    __shared__ float gauss[16];   // Shared memory for Gaussian weights

    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Initialize histogram
    for (int i = 0; i < 13; i++)
        hist[i * 32 + tx] = 0.0f;
    __syncthreads();

    // Precompute Gaussian weights
    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    if (tx < 15)
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7)); // Gaussian calculation

    int xp = (int)(d_Sift[bx + 0 * maxPts] - 6.5f);
    int yp = (int)(d_Sift[bx + 1 * maxPts] - 6.5f);
    int px = xp & 15;
    int x = tx - px;

    // Load data into shared memory
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = xp + x;
        int yi = yp + y;
        xi = max(0, min(xi, w - 1));
        yi = max(0, min(yi, h - 1));
        if (x >= 0 && x < 15) 
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    // Compute gradient and vote into histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1]  - data[memPos - 1];
            int bin = __float2int_rd(16.0f * atan2f(dy, dx) / 3.1416f + 16.5f) % 32; // Use modulo for safety
            float grad = sqrtf(dx * dx + dy * dy);
            atomicAdd(&hist[32 * (x - 1) + bin], grad * gauss[x] * gauss[y]);
        }
    }
    __syncthreads();

    // Reduce histogram across y
    if (tx < 32) {
        for (int y = 0; y < 5; y++)
            hist[y * 32 + tx] += hist[(y + 8) * 32 + tx];
        __syncthreads();
        for (int y = 0; y < 4; y++)
            hist[y * 32 + tx] += hist[(y + 4) * 32 + tx];
        __syncthreads();
        for (int y = 0; y < 2; y++)
            hist[y * 32 + tx] += hist[(y + 2) * 32 + tx];
        __syncthreads();

        // Smooth histogram
        float v = hist[tx];
        hist[tx] += hist[32 + tx];
        __syncthreads();

        if (tx < 32) { // Smoothing loop amend
            float smoothed = 6 * v + 4 * (hist[(tx + 1) & 31] + hist[(tx + 31) & 31]);
            smoothed += hist[(tx + 2) & 31] + hist[(tx + 30) & 31];
            hist[32 + tx] = smoothed; // Consolidate smoothing
        }
    }
    __syncthreads();

    // Find peaks in histogram
    if (tx == 0) {
        float maxval1 = 0.0f;
        float maxval2 = 0.0f;
        int i1 = -1, i2 = -1;
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }
        
        // Interpolation for the best peak location
        float valL = hist[(i1 + 31) % 32 + 32];
        float valR = hist[(i1 + 1) % 32 + 32];
        float peak = i1 + 0.5f * (valR - valL) / (2.0f * maxval1 - valL - valR);
        d_Sift[bx + 5 * maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        
        if (maxval2 < 0.8f * maxval1)
            i2 = -1;
        if (i2 >= 0) {
            valL = hist[(i2 + 31) % 32 + 32];
            valR = hist[(i2 + 1) % 32 + 32];
            peak = i2 + 0.5f * (valR - valL) / (2.0f * maxval2 - valL - valR);
            d_Sift[bx + 6 * maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        } else
            d_Sift[bx + 6 * maxPts] = i2;
    }
}