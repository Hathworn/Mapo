#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim.x for better readability
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Use blockDim.y for better readability
    int p = y * width + x; // Simplify index calculation

    if (x < width && y < height) // Conditionally execute only if within bounds
    {
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
    // Remove unnecessary __syncthreads() as it is not required here
}