#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx; // Simplified with standard multiplication
    const int y = blockIdx.y * 16;       // Simplified with standard multiplication
    const int b = blockDim.x;
    int p = y * width + x;               // Simplified with standard multiplication
    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        // Prevent out-of-bounds memory access
        minvals[tx] = INFINITY;   // Use INFINITY to handle min finding
        maxvals[tx] = -INFINITY;  // Use -INFINITY to handle max finding
    }
    for (int ty = 1; ty < 16; ty++) {
        p += width;
        if (x < width) {          // Proper condition check
            float val = d_Data[p];
            if (val < minvals[tx])
                minvals[tx] = val;
            if (val > maxvals[tx])
                maxvals[tx] = val;
        }
    }
    __syncthreads();
    for (int d = b / 2; d > 0; d >>= 1) { // Optimized loop for reduction
        if (tx < d) {
            if (minvals[tx + d] < minvals[tx])
                minvals[tx] = minvals[tx + d];
            if (maxvals[tx + d] > maxvals[tx])
                maxvals[tx] = maxvals[tx + d];
        }
        __syncthreads();
    }
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x); // Simplified index calculation
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}