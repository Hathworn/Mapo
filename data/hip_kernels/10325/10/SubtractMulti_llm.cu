#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Calculate the global x and y index for the thread
    const int x = blockIdx.x * SUBTRACTM_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACTM_H + threadIdx.y;
    const int z = threadIdx.z;
    
    // Pre-calculate size and check bounds
    int sz = height * pitch;
    int p = z * sz + y * pitch + x;
    
    if (x < width && y < height) {
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
    
    // Remove unnecessary __syncthreads(); it's redundant when no shared memory is used
}