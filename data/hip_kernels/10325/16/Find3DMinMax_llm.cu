#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int pitch, int height) {
    // Data cache
    __shared__ float data1[3 * (MINMAX_W + 2)];
    __shared__ float data2[3 * (MINMAX_W + 2)];
    __shared__ float data3[3 * (MINMAX_W + 2)];
    __shared__ float ymin1[(MINMAX_W + 2)];
    __shared__ float ymin2[(MINMAX_W + 2)];
    __shared__ float ymin3[(MINMAX_W + 2)];
    __shared__ float ymax1[(MINMAX_W + 2)];
    __shared__ float ymax2[(MINMAX_W + 2)];
    __shared__ float ymax3[(MINMAX_W + 2)];
    
    // Simplified index calculations
    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_W;
    const int xEndClamped = min(xStart + MINMAX_W - 1, width - 1);
    const int xReadPos = xStart + tx - WARP_SIZE;
    const int xWritePos = xStart + tx;
    
    int yq = 0;
    unsigned int output = 0;
    
    for (int y = 0; y < 34; y++) {
        output >>= 1;
        int memPos = yq * (MINMAX_W + 2) + (tx - WARP_SIZE + 1);
        int yp = min(max(32 * blockIdx.y + y - 1, 0), height - 1);
        int readStart = yp * pitch;

        // Optimize: Merge conditions to reduce branching
        if (tx >= (WARP_SIZE - 1) && xReadPos >= 0 && xReadPos < width) {
            data1[memPos] = d_Data1[readStart + xReadPos];
            data2[memPos] = d_Data2[readStart + xReadPos];
            data3[memPos] = d_Data3[readStart + xReadPos];
        } else if (tx >= (WARP_SIZE - 1)) {
            data1[memPos] = data2[memPos] = data3[memPos] = 0;
        }
        __syncthreads();

        int memPos2 = yq * (MINMAX_W + 2) + tx;
        if (y > 1 && tx < (MINMAX_W + 2)) {
            // Consolidated min/max calculations
            float min1 = fminf(fminf(data1[memPos], data1[yq * (MINMAX_W + 2) + (tx - WARP_SIZE + 1)]), data1[memPos2]);
            float min2 = fminf(fminf(data2[memPos], data2[yq * (MINMAX_W + 2) + (tx - WARP_SIZE + 1)]), data2[memPos2]);
            float min3 = fminf(fminf(data3[memPos], data3[yq * (MINMAX_W + 2) + (tx - WARP_SIZE + 1)]), data3[memPos2]);
            float max1 = fmaxf(fmaxf(data1[memPos], data1[yq * (MINMAX_W + 2) + (tx - WARP_SIZE + 1)]), data1[memPos2]);
            float max2 = fmaxf(fmaxf(data2[memPos], data2[yq * (MINMAX_W + 2) + (tx - WARP_SIZE + 1)]), data2[memPos2]);
            float max3 = fmaxf(fmaxf(data3[memPos], data3[yq * (MINMAX_W + 2) + (tx - WARP_SIZE + 1)]), data3[memPos2]);

            ymin1[tx] = min1;
            ymin2[tx] = fminf(fminf(min1, min2), min3);
            ymin3[tx] = min3;
            ymax1[tx] = max1;
            ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
            ymax3[tx] = max3;
        }
        __syncthreads();

        if (y > 1 && tx < MINMAX_W && xWritePos <= xEndClamped) {
            // Reduced load operations
            float minv = fminf(fminf(fminf(fminf(fminf(ymin2[tx], ymin2[tx + 2]), ymin1[tx + 1]), ymin3[tx + 1]), data2[memPos2 + tx]), data2[memPos2 + tx]);
            minv = fminf(minv, d_Threshold[1]);
            float maxv = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(ymax2[tx], ymax2[tx + 2]), ymax1[tx + 1]), ymax3[tx + 1]), data2[memPos2 + tx]), data2[memPos2 + tx]);
            maxv = fmaxf(maxv, d_Threshold[0]);
            if (data2[yq * (MINMAX_W + 2) + (tx - WARP_SIZE + 1) + 1] < minv || data2[yq * (MINMAX_W + 2) + (tx - WARP_SIZE + 1) + 1] > maxv) {
                output |= 0x80000000;
            }
        }
        __syncthreads();

        yq = (yq < 2 ? yq + 1 : 0);
    }

    if (tx < MINMAX_W && xWritePos < width) {
        int writeStart = blockIdx.y * pitch + xWritePos;
        d_Result[writeStart] = output;
    }
}