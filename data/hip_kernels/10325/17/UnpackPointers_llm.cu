#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    const int blockSize = 32; // Define block size
    int numPts = 0;
    
    // Use tidy loops and shared memory when necessary
    for (int y = 0; y < h / blockSize; y++) {
        for (int x = 0; x < w; x += 16) {
            unsigned int val = minmax[y * w + x + tx];
            
            // Check if there are points to process
            if (val && numPts < maxPts) {
                #pragma unroll // Use loop unrolling for efficiency
                for (int k = 0; k < blockSize; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = (y * blockSize + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}