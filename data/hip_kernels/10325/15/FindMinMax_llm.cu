#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    int p = y * pitch + x;

    // Initialize shared memory with data
    if (x < width && p < pitch * height) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        minvals[tx] = FLT_MAX; // Edge case: initialize to max possible
        maxvals[tx] = -FLT_MAX; // Edge case: initialize to min possible
    }

    // Iterate over the rest of the rows
    for (int ty = 1; ty < 16 && y + ty < height; ty++) {
        p += pitch;
        if (x < width) {
            float val = d_Data[p];
            minvals[tx] = min(minvals[tx], val);
            maxvals[tx] = max(maxvals[tx], val);
        }
    }
    __syncthreads();

    // Reduce min and max values within the block
    for (int d = blockDim.x / 2; d > 0; d >>= 1) {
        if (tx < d) {
            minvals[tx] = min(minvals[tx], minvals[tx + d]);
            maxvals[tx] = max(maxvals[tx], maxvals[tx + d]);
        }
        __syncthreads();
    }

    // Write results to global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}