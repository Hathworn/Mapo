#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;
    const int sharedHeight = h / 32;
    const int blockWidth = w / 16;
    
    for (int y = 0; y < sharedHeight; y++) {
        for (int bx = 0; bx < blockWidth; bx++) {
            int x = bx * 16;
            unsigned int val = minmax[y * w + x + tx];
            if (val) {
                #pragma unroll // Unroll inner loop for the constant range
                for (int k = 0; k < 32; k++) {
                    if ((val & 0x1) && (numPts < maxPts)) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1; // Shift value by one bit
                }
            }
        }
    }
}