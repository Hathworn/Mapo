#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    // Calculate the global thread ID in a linear fashion for better memory access pattern
    int tid = blockIdx.y * blockDim.y * pitch + blockIdx.x * blockDim.x + threadIdx.y * pitch + threadIdx.x;
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure thread is within bounds
    if (x < width && y < height) {
        d_Result[tid] = d_Data1[tid] - d_Data2[tid];
    }
    // __syncthreads() not necessary here because threads don't depend on each other
}