#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use blockIdx and threadIdx to ensure each thread processes a unique element
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        // Perform division only if index is within bounds
        accuracy[idx] /= N;
    }
}