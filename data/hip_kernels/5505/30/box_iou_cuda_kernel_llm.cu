#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void box_iou_cuda_kernel(float * box_iou, float4 * box1, float4 * box2, long M, long N, int idxJump) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= M * N) return; // Early exit if index is out of bounds

    size_t b1_idx = idx / N;
    size_t b2_idx = idx % N;

    float4 b1 = box1[b1_idx];
    float4 b2 = box2[b2_idx];

    float xmin1 = b1.x, ymin1 = b1.y, xmax1 = b1.z, ymax1 = b1.w;
    float xmin2 = b2.x, ymin2 = b2.y, xmax2 = b2.z, ymax2 = b2.w;

    // Compute inter-area dimensions
    float x_tl = fmaxf(xmin1, xmin2);
    float y_tl = fmaxf(ymin1, ymin2);
    float x_br = fminf(xmax1, xmax2);
    float y_br = fminf(ymax1, ymax2);

    float w = fmaxf(0.0f, x_br - x_tl + 1); // Width of the intersection area
    float h = fmaxf(0.0f, y_br - y_tl + 1); // Height of the intersection area

    float inter = w * h;
    float area1 = (xmax1 - xmin1 + 1) * (ymax1 - ymin1 + 1);
    float area2 = (xmax2 - xmin2 + 1) * (ymax2 - ymin2 + 1);
    float iou = inter / (area1 + area2 - inter);

    box_iou[b1_idx * N + b2_idx] = iou;
}