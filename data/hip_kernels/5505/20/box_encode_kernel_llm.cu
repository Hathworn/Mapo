#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void box_encode_kernel(float * targets_dx, float * targets_dy, float * targets_dw, float * targets_dh, float4 * boxes, float4 * anchors, float wx, float wy, float ww, float wh, size_t gt, size_t idxJump) {

    // Calculate global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Process elements in stride of idxJump
    for (int i = idx; i < gt; i += idxJump) {
        // Load data
        float4 curr_anchor = anchors[i];
        float4 curr_box = boxes[i];

        // Pre-compute values for transformation
        float ex_w = curr_anchor.z - curr_anchor.x + 1.0f;
        float ex_h = curr_anchor.w - curr_anchor.y + 1.0f;
        float ex_ctr_x = curr_anchor.x + 0.5f * ex_w;
        float ex_ctr_y = curr_anchor.y + 0.5f * ex_h;

        float gt_w = curr_box.z - curr_box.x + 1.0f;
        float gt_h = curr_box.w - curr_box.y + 1.0f;
        float gt_ctr_x = curr_box.x + 0.5f * gt_w;
        float gt_ctr_y = curr_box.y + 0.5f * gt_h;

        // Compute targets
        targets_dx[i] = wx * (gt_ctr_x - ex_ctr_x) / ex_w;
        targets_dy[i] = wy * (gt_ctr_y - ex_ctr_y) / ex_h;
        targets_dw[i] = ww * logf(gt_w / ex_w);
        targets_dh[i] = wh * logf(gt_h / ex_h);
    }
}