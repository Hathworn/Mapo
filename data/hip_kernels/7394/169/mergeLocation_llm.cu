#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (ptidx < npoints)
    {
        // Read location once and use shared variables to improve memory access efficiency
        const short2 loc = loc_[ptidx];
        const float scaled_x = loc.x * scale;
        const float scaled_y = loc.y * scale;

        x[ptidx] = scaled_x;
        y[ptidx] = scaled_y;
    }
}