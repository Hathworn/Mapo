#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Compute thread coordinates
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary check
    if (i >= h || j >= w) return;

    // Calculate flow and image row offsets
    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Get flow values
    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Calculate target coordinates
    float cx = u_ * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v_ * time_scale + static_cast<float>(i) + 1.0f;

    // Convert to integer coordinates
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Read source value
    float value = src[image_row_offset + j];

    // Check bounds and add to destination
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        atomicAdd(&dst[ty * image_stride + tx], value); // Use HIP atomicAdd
    }
}