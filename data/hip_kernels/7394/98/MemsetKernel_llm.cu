#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds
    if (i < h && j < w)
    {
        int pos = i * w + j;
        image[pos] = value; // Set image value
    }
}