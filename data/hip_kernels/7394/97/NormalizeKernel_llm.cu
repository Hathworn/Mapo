#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Return early if the thread is out of image bounds
    if (i >= h || j >= w) return;

    const int pos = i * s + j;
    float scale = normalization_factor[pos];
    
    // Use a ternary operator for conditionally computing inverse scale
    float invScale = (scale != 0.0f) ? 1.0f / scale : 1.0f;

    // Multiply image by inverse scale
    image[pos] *= invScale;
}