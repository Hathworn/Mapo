#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use an atomic operation to ensure thread-safe access to the accuracy variable
    if (threadIdx.x == 0) {
        atomicExch(accuracy, *accuracy / N);
    }
}