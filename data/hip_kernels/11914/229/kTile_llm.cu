#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate total number of threads
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    // Use a stride loop for better coalesced memory access
    for (uint i = idx; i < tgtWidth * tgtHeight; i += stride) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Optimize memory access for improved performance
        __syncthreads();
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}