#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    // Advance pointers for the current batch element
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    // Thread index
    const int idx = threadIdx.x;
    // Shared memory buffer for reduction
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    // Per-thread accumulator
    float tmp = 0;

    // First level of reduction: each thread computes partial sum
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    
    // Store partial results in shared memory
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Reduce within the block to calculate the final sum
    if (idx == 0) {
        float sum = 0;
        for (int i = 0; i < blockDim.x; ++i) {
            sum += reduction_buffer[i];
        }
        reduction_buffer[0] = sum;
    }
    __syncthreads();

    // Scalar value for all threads in the block
    const float final_sum = reduction_buffer[0];

    // Compute gradient: each thread updates multiple elements
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - final_sum);
    }
}