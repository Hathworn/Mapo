#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];
    
    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;
    
    // Use loop unrolling for efficiency
    for (uint i = eidx; i < numElements; i += gridDim.x * DP_BLOCKSIZE) {
        sum += a[i] * b[i];
    }
    
    shmem[threadIdx.x] = sum;
    __syncthreads();
    
    // Optimize reduction using loop unrolling
    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    __syncthreads();
    
    if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    __syncthreads();
    
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
        if (threadIdx.x == 0) {
            target[blockIdx.x] = mysh[0];
        }
    }
}