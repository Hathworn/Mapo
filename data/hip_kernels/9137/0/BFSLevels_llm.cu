#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize BFS kernel function
__global__ void BFSLevels(int *vertices, int *edges, int *distances, int *predecessors, int *vertIndices, int *edgeSize, bool *levels, bool *visitedVertices, bool *foundDest, int numVert, int destination) 
{
    int thrID = threadIdx.x + blockIdx.x * blockDim.x;

    extern __shared__ bool sharedDestFound[];
    bool localDestFound = false;

    if (thrID < numVert && !*foundDest) 
    {
        int curVert = vertices[thrID];

        if (levels[curVert]) 
        {
            levels[curVert] = false;
            visitedVertices[curVert] = true;

            int edgesEnd = edgeSize[thrID] + vertIndices[thrID];

            for (int edgeIter = vertIndices[thrID]; edgeIter < edgesEnd; ++edgeIter) 
            {
                int nextVert = edges[edgeIter];

                if (!visitedVertices[nextVert]) 
                {
                    distances[nextVert] = distances[curVert] + 1;
                    levels[nextVert] = true;
                    predecessors[nextVert] = curVert;

                    if (nextVert == destination) 
                    {
                        *foundDest = true;
                        localDestFound = true;
                        break; // Exit the loop as destination found
                    }
                }
            }
        }
        
        // Update shared memory only from a single thread (block-level synchronization)
        if (localDestFound) {
            sharedDestFound[0] = true;
        }
    }

    // Sync and update the global flag from shared
    __syncthreads();
    if (sharedDestFound[0]) {
        *foundDest = true;
    }
}