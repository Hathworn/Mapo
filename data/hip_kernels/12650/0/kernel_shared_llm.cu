#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define FIBER 32
#define MATRIX_SIZE 2048
#define DATA_SIZE MATRIX_SIZE * MATRIX_SIZE * sizeof(int)
#define MAX_MATRIX_SIZE (MATRIX_SIZE * MATRIX_SIZE)

using namespace std;

__global__ void kernel_shared(int *A, int *C, int *B, int *result) {
    __shared__ int shared_memory[FIBER][FIBER];

    int global_i = blockIdx.y * blockDim.y + threadIdx.y; // Optimized index calculation
    int global_j = blockIdx.x * blockDim.x + threadIdx.x; // Optimized index calculation

    shared_memory[threadIdx.y][threadIdx.x] = B[global_i * MATRIX_SIZE + global_j];

    __syncthreads();

    int first_index = global_i * MATRIX_SIZE + global_j; // Optimized indexing order
    int second_index = global_j * MATRIX_SIZE + global_i; // Optimized indexing order

    if (first_index < MAX_MATRIX_SIZE && second_index < MAX_MATRIX_SIZE) {
        result[first_index] = (A[first_index] + A[first_index]) * shared_memory[threadIdx.y][threadIdx.x] - C[first_index];
    }
}