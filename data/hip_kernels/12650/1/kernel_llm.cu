#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define FIBER 32
#define MATRIX_SIZE 2048
#define DATA_SIZE MATRIX_SIZE * MATRIX_SIZE * sizeof(int)
#define MAX_MATRIX_SIZE (MATRIX_SIZE * MATRIX_SIZE)

using namespace std;

__global__ void kernel(int *A, int *C, int *B, int *result) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Consolidate index calculations
    int first_index = i + j * MATRIX_SIZE;
    
    if (first_index < MAX_MATRIX_SIZE) {
        // Compute second_index only when needed
        int second_index = j + i * MATRIX_SIZE;
        if (second_index < MAX_MATRIX_SIZE) {
            int valueA = A[first_index];
            result[first_index] = (valueA + valueA) * B[second_index] - C[first_index];
        }
    }
}