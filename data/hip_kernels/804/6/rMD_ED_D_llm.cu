#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rMD_ED_D(float *S, float *T, int window_size, int dimensions, float *data_out, int trainSize, int gm) {

    long long int i, j, p;
    float sumErr = 0, dd = 0;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= (trainSize * window_size)) 
        return;
    
    extern __shared__ float T2[];

    // offset training set
    int s = dimensions * 2 * window_size * (idx / window_size);
    int t = s + idx % window_size;

    // Pre-load T into shared memory if gm == 0
    if (gm == 0) {
        if (threadIdx.x == 0) {
            for (i = 0; i < dimensions; i++) {
                for (j = 0; j < window_size; j++) {
                    T2[window_size * i + j] = T[window_size * i + j];
                }
            }
        }
        __syncthreads();
    }

    for (j = 0; j < window_size; j++) {
        dd = 0;
        for (p = 0; p < dimensions; p++) {
            float diff = S[(t + p * 2 * window_size) + j] - (gm == 0 ? T2[(p * window_size) + j] : T[(p * window_size) + j]);
            dd += diff * diff;
        }
        sumErr += dd;
    }
    data_out[idx] = sqrt(sumErr);
}