#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**
* @brief cudaCreateBuffer Allocates a cuda buffer and stops the programm on error.
* @param size
* @return
*/
__global__ void kernelSetDoubleBuffer(float* gpuBuffPtr, float v, size_t size)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check index within bounds and assign value
    if (index < size) {
        gpuBuffPtr[index] = v;
    }
}