#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelNormalizeMotionEnergyAsync(int bsx, int bsy, int n, float alphaPNorm, float alphaQNorm, float betaNorm, float sigmaNorm, float* gpuEnergyBuffer)
{
    int bufferPos = threadIdx.x + blockIdx.x * blockDim.x;
    float sigmaNorm2_2 = 2 * sigmaNorm * sigmaNorm;
    float gausCoe = 1 / (sigmaNorm2_2 * M_PI);

    // Declare shared memory for local block processing
    __shared__ float sharedMem[BLOCK_SIZE + 4]; // Assuming BLOCK_SIZE is defined

    if (bufferPos < n) {
        int bx, by;
        int bxy = bufferPos / (bsx * bsy);
        bx = bxy % bsx;
        by = bxy / bsx;
        
        // Load data into shared memory
        sharedMem[threadIdx.x] = gpuEnergyBuffer[bufferPos];
        __syncthreads();

        // Read energy
        float I = sharedMem[threadIdx.x];
        float q_i = 0;

        // Normalize over 5x5 region
        for (int y = -2; y <= 2; y++) {
            int by_ = by + y;

            if (by_ < 0 || by_ >= bsy)
                continue;

            for (int x = -2; x <= 2; x++) {
                int bx_ = bx + x;

                if (bx_ < 0 || bx_ >= bsx || (bx == bx_ && by == by_))
                    continue;

                // Use shared memory to avoid extra global memory access
                float gaus = gausCoe * expf(-(bx_ * bx_ + by_ * by_) / sigmaNorm2_2);
                int sharedIdx = (by_ % blockDim.y) * blockDim.x + (bx_ % blockDim.x);

                if (sharedIdx < BLOCK_SIZE + 4) { // Ensure no out of bounds access
                    q_i += sharedMem[sharedIdx] * gaus;
                }
            }
        }
        q_i /= alphaQNorm;

        // Compute p_i
        float p_i = (I * betaNorm) / (alphaPNorm + I + q_i);

        // Use normalized value
        gpuEnergyBuffer[bufferPos] = p_i;
    }
}