#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelProcessEventsBatchAsync(uint8_t* gpuEventsX,uint8_t* gpuEventsY,int gpuEventListSize, float* gpuFilter, int fsx, int fsy, int fsz, float* gpuBuffer, int ringBufferIdx, int bsx, int bsy, int bsz, int fs_xy, int fn)
{
    // Calculate filter idx
    int filterPos = threadIdx.x + blockIdx.x * blockDim.x;

    // Per block shared memory
    __shared__ uint8_t gpuEventListSharedX[MAX_SHARED_GPU_EVENTS];
    __shared__ uint8_t gpuEventListSharedY[MAX_SHARED_GPU_EVENTS];

    // How many runs do we need to process all events
    int processingRuns = (gpuEventListSize + MAX_SHARED_GPU_EVENTS - 1) / MAX_SHARED_GPU_EVENTS;
    
    // Events for each thread to read
    int eventReadsPerThread = (MAX_SHARED_GPU_EVENTS + blockDim.x - 1) / blockDim.x;

    // Offset in global event buffer
    int globalEventIdx = threadIdx.x;

    // Idx valid
    if (filterPos < fn) {
        // Read filter coefficient from global memory
        float filterVal = gpuFilter[filterPos];

        // Compute x, y, z coordinates in buffer
        int fz = filterPos / fs_xy;
        int fxy = filterPos % fs_xy;
        int fy = fxy / fsx;
        int fx = fxy % fsx;

        // Convert buffer coordinates (mirror all axes -> convolution instead of correlation)
        int bz = ((ringBufferIdx + (fsz - 1) - fz ) % bsz);
        int bx_tmp = fsx / 2 - fx;
        int by_tmp = fsy / 2 - fy;
        int bPos_tmp = bz * bsy * bsx;

        int sharedEventCnt = MAX_SHARED_GPU_EVENTS;

        // Iterate over event list in blocks, stored in shared memory
        for(int runIdx = 0; runIdx < processingRuns; runIdx++) {
            // Last run? Compute size of shared event list
            if(runIdx + 1 == processingRuns) {
                sharedEventCnt = gpuEventListSize % MAX_SHARED_GPU_EVENTS;
            }

            // Compute index in shared memory
            int localEventIdx = threadIdx.x;

            // Fill the shared memory either with MAX_SHARED_GPU_EVENTS
            for(int i = 0; i < eventReadsPerThread; i++) {
                // Valid indices
                if(globalEventIdx < gpuEventListSize) {
                    // Load event into shared memory
                    if(localEventIdx < sharedEventCnt) {
                        gpuEventListSharedX[localEventIdx] = gpuEventsX[globalEventIdx];
                        gpuEventListSharedY[localEventIdx] = gpuEventsY[globalEventIdx];
                    }
                }
                // Goto next event for which this thread is responsible
                localEventIdx += blockDim.x;
                globalEventIdx += blockDim.x;
            }

            // Synchronize threads and wait until shared memory is filled
            __syncthreads();

            // Iterate over every event block in shared memory
            for(localEventIdx = 0; localEventIdx < sharedEventCnt; localEventIdx++) {
                // Compute corresponding buffer coordinate
                int bx = bx_tmp + gpuEventListSharedX[localEventIdx];
                int by = by_tmp + gpuEventListSharedY[localEventIdx];

                // Check for valid buffer position (flip buffer z)
                if(bx >= 0 && bx < bsx && by >= 0 && by < bsy) {
                    int bufferPos = bPos_tmp + by * bsx + bx;
                    // Add each filter coefficient to the global buffer
                    atomicAdd(gpuBuffer + bufferPos, filterVal);
                }
            }
            __syncthreads();  // Ensures completion of processing before next iteration
        }
    }
}