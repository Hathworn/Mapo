#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelReadMotionEnergyAsync(float* gpuConvBufferl1, float* gpuConvBufferl2, int ringBufferIdx, int bsx, int bsy, int n, float* gpuEnergyBuffer)
{
    int bufferPos = threadIdx.x + blockIdx.x * blockDim.x;
    if(bufferPos < n) {
        // Precompute the constant index offset outside the loop
        int offset = ringBufferIdx * bsx * bsy;
        int bufferPosConv = bufferPos + offset;

        // Access memory once and store in registers
        float l1 = gpuConvBufferl1[bufferPosConv];
        float l2 = gpuConvBufferl2[bufferPosConv];

        // Use the fmaf function for optimized computation (fmaf(a, b, c) computes a * b + c)
        gpuEnergyBuffer[bufferPos] = sqrtf(fmaf(l1, l1, l2 * l2));
    }
}