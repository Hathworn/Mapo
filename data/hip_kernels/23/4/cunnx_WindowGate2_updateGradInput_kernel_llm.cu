#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_WindowGate2_updateGradInput_kernel(float *gradInput, float *error, float* targetCentroids, 
                                                         const float *centroids, const float *input, 
                                                         const float *inputIndice, const float *outputIndice, 
                                                         const float* output, const float* gradOutput, 
                                                         int inputSize, int outputSize, int inputWindowSize, 
                                                         int outputWindowSize, int windowStride, float c, 
                                                         float d, float e, float lr) {
    unsigned int tx = threadIdx.x;
    unsigned int k = blockIdx.x;
    const float *gradOutput_k = gradOutput + outputWindowSize * k;
    float *gradInput_k = gradInput + inputSize * k;

    float *gradInputWindow = gradInput_k + (int)(inputIndice[k] - 1);

    // Use the shared memory to optimize global memory access
    extern __shared__ float sharedGradOutputChannel[];
    
    for (int i = tx; i < inputWindowSize; i += blockDim.x) {
        float sum = 0;

        // Load gradOutputChannel into shared memory in chunks
        for (int j = threadIdx.x; j < windowStride; j+=blockDim.x) {
            sharedGradOutputChannel[j] = gradOutput_k[i * windowStride + j];
        }
        __syncthreads();

        // Sum over shared memory to reduce global memory access
        for (int j = 0; j < windowStride; j++) {
            sum += sharedGradOutputChannel[j];
        }

        gradInputWindow[i] += sum;
    }
}