#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunnx_BlockSparse_updateOutput_kernel( float *output, const float *input, const float *outputIndice, const float *outputScale, const float *bias, int outputSize, int nOutputBlock, int inputWindowSize, int outputWindowSize)
{
    extern __shared__ float buffer[];  // Use dynamic shared memory
    int tx = threadIdx.x;
    int k = blockIdx.x;
    
    float *output_k = output + k * outputWindowSize * outputSize;
    const float *input_k = input + k * inputWindowSize * outputWindowSize * outputSize;
    const float *outputIndice_k = outputIndice + k * outputWindowSize;
    const float *outputScale_k = outputScale + k * outputWindowSize;

    for (int m = 0; m < outputWindowSize; ++m) {
        int outputIdx = static_cast<int>(outputIndice_k[m]) - 1;
        float outputScale = outputScale_k[m];

        for (int j = tx; j < outputSize; j += blockDim.x) {
            // Efficiently load bias
            float tempOutput = bias[outputIdx * outputSize + j];

            // Unroll the loop for efficiency
            for (int l = 0; l < inputWindowSize; ++l) {
                tempOutput += input_k[l * outputWindowSize * outputSize + m * outputSize + j];
            }
            
            // Store the scaled output to global memory
            output_k[m * outputSize + j] = outputScale * tempOutput;
        }
    }
}