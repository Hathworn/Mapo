#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunnx_LazyKBest_updateGradInput_kernel(float *gradInput, const float *indice, const float *gradOutput, int inputSize, int outputSize)
{
    // Use shared memory for better cache locality
    __shared__ float sharedGradOutput[1024];  // Adjust size according to outputSize if necessary
    __shared__ int sharedIndice[1024];        // Adjust size according to outputSize if necessary

    int tx = threadIdx.x;
    int k = blockIdx.x;
    if (tx < outputSize) {
        sharedGradOutput[tx] = gradOutput[k * outputSize + tx];
        sharedIndice[tx] = indice[k * outputSize + tx] - 1;  // Pre-compute -1
    }
    __syncthreads();  // Sync threads after loading to shared memory

    float *gradInput_k = gradInput + k * inputSize;

    // Reduce step size for potential unrolling, processing multiple elements per thread
    for (int i = tx; i < outputSize; i += blockDim.x * 2) {
        gradInput_k[sharedIndice[i]] = sharedGradOutput[i];
        if (i + blockDim.x < outputSize) {  // Check boundary for second element
            gradInput_k[sharedIndice[i + blockDim.x]] = sharedGradOutput[i + blockDim.x];
        }
    }
}