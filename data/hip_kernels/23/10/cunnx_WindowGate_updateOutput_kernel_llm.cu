#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunnx_WindowGate_updateOutput_kernel(float *output, float *centroids, float *normalizedCentroids, float *outputIndice, const float *input, const float *noise, int inputSize, int outputSize, int outputWindowSize, float a, float b, int train)
{
    __shared__ float buffer[WINDOWGATE_THREADS];
    unsigned int tx = threadIdx.x;
    unsigned int k = blockIdx.x;
    const float *input_k = input + inputSize * k;
    float *output_k = output + outputWindowSize * k;

    // Initialize buffer for reduction
    buffer[tx] = 0.0f;

    // Compute sum of weighted indices
    for (unsigned int i = tx; i < inputSize; i += blockDim.x)
        buffer[tx] += input_k[i] * (float)(i + 1);

    // Perform reduction within thread block
    for (unsigned int stride = WINDOWGATE_THREADS >> 1; stride > 0; stride >>= 1)
    {
        __syncthreads();
        if (tx < stride)
            buffer[tx] += buffer[tx + stride];
    }

    if (tx == 0)
    {
        float centroid = buffer[0];

        // Normalize centroid between 0 and 1
        centroid /= (float)inputSize;
        normalizedCentroids[k] = centroid;

        // Add noise and clamp centroid if training
        if (train)
        {
            centroid += noise[k];
            centroid = fminf(fmaxf(0.0f, centroid), 1.0f);
        }

        // Align centroid to output space
        centroid *= (float)outputSize;

        float outputIdx = centroid - 0.5f * (float)outputWindowSize;

        // Clip output index to valid range
        outputIdx = fminf(outputIdx, outputSize - outputWindowSize + 1);
        outputIdx = fmaxf(outputIdx, 1.0f);

        outputIdx = ceilf(outputIdx);

        // Align centroid to output window
        centroid -= (outputIdx - 1);

        outputIndice[k] = (int)outputIdx;
        centroids[k] = centroid;
        buffer[0] = centroid;
    }

    __syncthreads();

    float centroid = buffer[0];

    // Apply Gaussian blur
    for (int i = tx; i < outputWindowSize; i += blockDim.x)
    {
        float x = (float)(i + 1) - centroid;
        output_k[i] = a * expf(x * x * b);
    }
}