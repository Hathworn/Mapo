#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunnx_WindowGate2_updateOutput_kernel(float *output, float *centroids, float *normalizedCentroids, float *inputIndice, float *outputIndice, const float *input, const float *noise, int inputSize, int outputSize, int inputWindowSize, int outputWindowSize, int windowStride, int train) {
    __shared__ float buffer[WINDOWGATE2_THREADS + 1]; // Shared memory buffer
    unsigned int tx = threadIdx.x;  // Thread index
    unsigned int k = blockIdx.x;    // Block index for batch processing
    const float *input_k = input + inputSize * k;  // Input segment for current batch
    float *output_k = output + outputWindowSize * k;  // Output segment for current batch

    // Initialize buffer to zero
    buffer[tx] = 0;
    // Calculate weighted sum of input indices multiplied by inputs
    for (unsigned int i = tx; i < inputSize; i += blockDim.x) {
        buffer[tx] += input_k[i] * (float)(i + 1);
    }

    // Perform parallel reduction within a block to accumulate the sum
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (tx < stride) {
            buffer[tx] += buffer[tx + stride];
        }
    }

    // Single thread computes final centroid from reduction result
    if (tx == 0) {
        float centroid = buffer[0];
        centroid /= (float)(inputSize);  // Normalize centroid to range [0, 1]
        normalizedCentroids[k] = centroid;

        if (train) {
            centroid += noise[k];  // Add noise during training
            centroid = fminf(fmaxf(0, centroid), 1);  // Clamp to [0, 1]
        }
        centroid *= (float)(outputSize); // Scale centroid to output size

        float inputIdx = centroid / (float)(inputSize) - 0.5f * inputWindowSize;
        float outputIdx = centroid - 0.5f * outputWindowSize;

        // Clamp the input and output indices
        inputIdx = fminf(inputIdx, inputSize - inputWindowSize + 1);
        inputIdx = fmaxf(inputIdx, 1);
        outputIdx = fminf(outputIdx, outputSize - outputWindowSize + 1);
        outputIdx = fmaxf(outputIdx, 1);

        inputIdx = ceilf(inputIdx);  // Round up to nearest whole number
        outputIdx = ceilf(outputIdx); // Round up to nearest whole number

        // Adjust centroid relative to aligned outputWindow
        centroid -= (outputIdx - 1);

        inputIndice[k] = (int)inputIdx;
        outputIndice[k] = (int)outputIdx;
        centroids[k] = centroid;

        buffer[WINDOWGATE2_THREADS] = inputIdx; // Store aligned input index
    }

    __syncthreads();

    float inputIdx = buffer[WINDOWGATE2_THREADS]; // Load aligned input index
    const float *inputWindow = input_k + (int)inputIdx; // Pointer to input window

    // Map input window to output window based on stride
    for (int i = tx; i < outputWindowSize; i += blockDim.x) {
        output_k[i] = inputWindow[(int)floorf((float)i / windowStride)];
    }
}