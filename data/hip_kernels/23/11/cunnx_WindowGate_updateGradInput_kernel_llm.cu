#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_WindowGate_updateGradInput_kernel( float *gradInput, float *error, float* targetCentroids, const float *centroids,const float *input, const float *outputIndice, const float* output, const float* gradOutput, int inputSize, int outputSize, int outputWindowSize, float c, float d, float e, float lr)
{
    __shared__ float buffer[WINDOWGATE_THREADS+1];
    unsigned int tx = threadIdx.x;
    unsigned int k = blockIdx.x;
    const float *gradOutput_k = gradOutput + outputWindowSize*k;
    const float *output_k = output + outputWindowSize*k;
    const float *input_k = input + inputSize*k;
    float *gradInput_k = gradInput + inputSize*k;
    float centroid = centroids[k];

    // Initialize shared memory for summation
    buffer[tx] = 0;
    float sum = 0;
    for (unsigned int i = tx; i < outputWindowSize; i += blockDim.x)
    {
        sum += gradOutput_k[i] * output_k[i] * ((float)(i + 1) - centroid);
    }
    buffer[tx] = sum;

    // Parallel reduction to sum over buffer
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
    {
        __syncthreads();
        if (tx < stride)
        {
            buffer[tx] += buffer[tx + stride];
        }
    }

    if (tx == 0)
    {
        int outputIdx = outputIndice[k];
        float gradCentroid = buffer[0] * c;
        centroid -= (lr * gradCentroid);
        centroid += outputIdx - 1;
        centroid /= (float)(outputSize);
        targetCentroids[k] = centroid;
        buffer[WINDOWGATE_THREADS] = centroid * (float)(inputSize);
    }

    __syncthreads();
    float targetCentroid = buffer[WINDOWGATE_THREADS];

    buffer[tx] = 0;
    // Compute target values and update gradInput
    for (int i = tx; i < inputSize; i += blockDim.x)
    {
        float target = (float)(i + 1) - targetCentroid;
        target = d * expf(target * target * e);
        float input = input_k[i];
        buffer[tx] -= logf(input + 1e-7f) * target;
        gradInput_k[i] = -target / (input + 1e-7f);
    }

    // Final reduction to compute error
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
    {
        __syncthreads();
        if (tx < stride)
        {
            buffer[tx] += buffer[tx + stride];
        }
    }

    if (tx == 0)
    {
        error[k] = buffer[tx];
    }
}