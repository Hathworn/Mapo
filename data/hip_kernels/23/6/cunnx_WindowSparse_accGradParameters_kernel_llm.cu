#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunnx_WindowSparse_accGradParameters_kernel( float *gradWeight, float* gradBias, float *gradOutput, float *input, float *inputIndice, float *outputIndice, int inputWindowSize, int outputWindowSize, int inputSize, int outputSize, float scale)
{
    __shared__ float buffer[WINDOWSPARSE_THREADS];
    int tx = threadIdx.x;
    int i_step = blockDim.x;
    int k = blockIdx.x;

    int inputIdx = (int)inputIndice[k] - 1;
    int outputIdx = (int)outputIndice[k] - 1;

    float *input_k = input + k*inputWindowSize;
    float *gradOutput_k = gradOutput + k*outputWindowSize;
    float *gradWeight_k = gradWeight + outputIdx*inputSize + inputIdx;
    float *gradBias_k = gradBias + outputIdx;

    // Load inputs and scale them in parallel
    for (int i=tx; i<inputWindowSize; i+=i_step) {
        buffer[tx] = input_k[i]*scale;
    }
    __syncthreads(); // Synchronize to ensure all threads have loaded inputs

    // Multiply accumulate weights in parallel
    for (int i=tx; i<inputWindowSize; i+=i_step) {
        for (int j=0; j<outputWindowSize; j++) {
            atomicAdd(&(gradWeight_k[j*inputSize + i]), gradOutput_k[j]*buffer[i]);
        }
    }

    // Multiply accumulate biases in parallel
    for (int j=tx; j<outputWindowSize; j+=i_step) {
        atomicAdd(&(gradBias_k[j]), gradOutput_k[j]*scale);
    }
}