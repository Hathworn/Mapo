#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread 0 of block 0 to update the value
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        *accuracy /= N;
    }
}