#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultiSoftmaxCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, float* top5Probs, const int numCases, const int numOut, const int setSize) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        // Use CUDA's fast math function
        labelLogProbs[tx] = __logf(labelp);

        int numBiggerProbs = 0, numEqualsProbs = 0;

        // Optimize memory access by accessing contiguous memory
        for (int i = 0; i < numOut; ++i) {
            float prob = probs[i * numCases + tx];
            numBiggerProbs += prob > labelp;
            numEqualsProbs += prob == labelp;
        }

        // Reduce computation inside the loop
        const int slotsLeft = setSize - numBiggerProbs;
        top5Probs[tx] = slotsLeft <= 0 ? 0.0f : (numEqualsProbs <= slotsLeft ? 1.0f : float(slotsLeft) / float(numEqualsProbs));
        
        correctProbs[tx] = (labelp != maxp) ? 0.0f : 1.0f / float(numEqualsProbs);
    }
}