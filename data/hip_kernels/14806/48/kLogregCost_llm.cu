#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    // Ensure thread works on a valid case
    if (tx < numCases) {
        const int label = __float2int_rd(labels[tx]); // Use fast rounding intrinsic
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Initialize numMax to count occurrences of the maximum probability
        int numMax = 0;
        bool isCorrectLabel = labelp == maxp; // Check if the label probabilty equals max probability

        // Loop unrolling to optimize counting max probability occurrences
        for (int i = 0; i < numOut; i += 4) {
            if (i < numOut) numMax += (probs[i * numCases + tx] == maxp);
            if (i + 1 < numOut) numMax += (probs[(i + 1) * numCases + tx] == maxp);
            if (i + 2 < numOut) numMax += (probs[(i + 2) * numCases + tx] == maxp);
            if (i + 3 < numOut) numMax += (probs[(i + 3) * numCases + tx] == maxp);
        }

        // Assign probability based on most probable label check
        correctProbs[tx] = isCorrectLabel ? 1.0f / float(numMax) : 0.0f;
    }
}