#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target,  const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Loop unrolling and avoiding branch divergence
    while (eidx < numElements) {
        sum += a[eidx] * b[eidx];
        eidx += gridDim.x * DP_BLOCKSIZE;
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Efficient reduction within a warp
    if (threadIdx.x < 256) { shmem[threadIdx.x] += shmem[threadIdx.x + 256]; } __syncthreads();
    if (threadIdx.x < 128) { shmem[threadIdx.x] += shmem[threadIdx.x + 128]; } __syncthreads();
    if (threadIdx.x < 64) { shmem[threadIdx.x] += shmem[threadIdx.x + 64]; } __syncthreads();

    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    // Only the first thread in the block writes to the target
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}