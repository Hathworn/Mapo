#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x; // Use blockDim.x for better readability and flexibility
    /* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(seed, tidx, 0, &state[tidx]); // streamlined comment
}