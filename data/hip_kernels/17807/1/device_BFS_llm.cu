#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_BFS(const int* edges, const int* dests, int* labels, int* visited, int* c_frontier_tail, int* c_frontier, int* p_frontier_tail, int* p_frontier) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check index within bounds
    if (index < *p_frontier_tail) {
        int c_vertex = p_frontier[index];
        
        // Preload vertex edges information to avoid recalculating it
        int start_edge = edges[c_vertex];
        int end_edge = edges[c_vertex + 1];

        // Loop through edges more efficiently
        for (int i = start_edge; i < end_edge; i++) {
            int destination_vertex = dests[i];
            
            // Use atomic operation efficiently
            if (atomicExch(visited + destination_vertex, 1) == 0) {
                int old_tail = atomicAdd(c_frontier_tail, 1);
                
                // Efficient memory access
                c_frontier[old_tail] = destination_vertex;
                labels[destination_vertex] = labels[c_vertex] + 1;
            }
        }
    }
}