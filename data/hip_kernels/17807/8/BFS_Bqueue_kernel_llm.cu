#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_Bqueue_kernel(unsigned int* p_frontier, unsigned int* p_frontier_tail, unsigned int* c_frontier, unsigned int* c_frontier_tail, unsigned int* edges, unsigned int* dest, unsigned int* label, unsigned int* visited) {
    __shared__ unsigned int c_frontier_s[BLOCK_QUEUE_SIZE];
    __shared__ unsigned int c_frontier_tail_s;
    __shared__ unsigned int our_c_frontier_tail;

    // Initialize the shared tail counter
    if (threadIdx.x == 0) {
        c_frontier_tail_s = 0;
    }
    __syncthreads();

    const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < *p_frontier_tail) {
        const unsigned int my_vertex = p_frontier[tid];
        for (unsigned int i = edges[my_vertex]; i < edges[my_vertex + 1]; ++i) {
            const unsigned int dest_vertex = dest[i];  // Cache dest computation
            const unsigned int was_visited = atomicExch(&(visited[dest_vertex]), 1);
            if (!was_visited) {
                label[dest_vertex] = label[my_vertex];
                const unsigned int my_tail = atomicAdd(&c_frontier_tail_s, 1);
                if (my_tail < BLOCK_QUEUE_SIZE) {
                    c_frontier_s[my_tail] = dest_vertex;
                } else {
                    // Ensure no overflow of local queue
                    c_frontier_tail_s = BLOCK_QUEUE_SIZE;
                    const unsigned int my_global_tail = atomicAdd(c_frontier_tail, 1);
                    c_frontier[my_global_tail] = dest_vertex;
                }
            }
        }
        __syncthreads();

        if (threadIdx.x == 0) {
            our_c_frontier_tail = atomicAdd(c_frontier_tail, c_frontier_tail_s);
        }
        __syncthreads();

        for (unsigned int i = threadIdx.x; i < c_frontier_tail_s; i += blockDim.x) {
            c_frontier[our_c_frontier_tail + i] = c_frontier_s[i];
        }
    }
}