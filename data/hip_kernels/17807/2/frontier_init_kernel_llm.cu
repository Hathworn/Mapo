#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void frontier_init_kernel(int* p_frontier_tail_d, int* c_frontier_tail_d, int* p_frontier_d, int* visited_d, int* label_d, int source) {
    // Obtain thread ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Only the first thread executes initialization
    if (tid == 0) {
        visited_d[source] = 1;
        *c_frontier_tail_d = 0;
        p_frontier_d[0] = source;
        *p_frontier_tail_d = 1;
        label_d[source] = 0;
    }
}