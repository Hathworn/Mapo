#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_UNIFIED(int source, int* edges, int* dest, int* label, int* visited, int *c_frontier_tail, int *c_frontier, int *p_frontier_tail, int *p_frontier)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < *p_frontier_tail)
    {
        int c_vertex = p_frontier[i];
        int start_edge = edges[c_vertex];
        int end_edge = edges[c_vertex + 1];
        
        // Loop unrolling for efficiency
        for (int j = start_edge; j < end_edge; ++j)
        {
            int was_visited = atomicExch(visited + dest[j], 1);
            if (!was_visited)
            {
                int old_tail = atomicAdd(c_frontier_tail, 1);
                c_frontier[old_tail] = dest[j];
                label[dest[j]] = label[c_vertex] + 1;
            }
        }
    }
}
```
