#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BFS_Bqueue_kernel(int* p_frontier, int* p_frontier_tail, int* c_frontier, int* c_frontier_tail, int* edges, int* dest, int* label, int* visited) {

    __shared__ int c_frontier_s[BLOCK_QUEUE_SIZE];
    __shared__ int c_frontier_tail_s;
    __shared__ int our_c_frontier_tail;

    // Initialize shared c_frontier_tail_s to 0 by the first thread
    if (threadIdx.x == 0) {
        c_frontier_tail_s = 0;
    }
    __syncthreads();

    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < *p_frontier_tail) {
        const int my_vertex = p_frontier[tid];
        for (int i = edges[my_vertex]; i < edges[my_vertex + 1]; ++i) {
            const int was_visited = atomicExch(&(visited[dest[i]]), 1);
            if (!was_visited) {
                label[dest[i]] = label[my_vertex] + 1;
                const int my_tail = atomicAdd(&c_frontier_tail_s, 1);
                
                // Use thread cooperation for shared memory update
                if (my_tail < BLOCK_QUEUE_SIZE) {
                    c_frontier_s[my_tail] = dest[i];
                } else {
                    const int my_global_tail = atomicAdd(c_frontier_tail, 1);
                    c_frontier[my_global_tail] = dest[i];
                }
            }
        }
        __syncthreads();

        // Only the first thread updates our_c_frontier_tail
        if (threadIdx.x == 0) {
            our_c_frontier_tail = atomicAdd(c_frontier_tail, c_frontier_tail_s);
        }
        __syncthreads();

        // Distribute work of copying from shared to global memory across threads
        for (int i = threadIdx.x; i < c_frontier_tail_s; i += blockDim.x) {
            c_frontier[our_c_frontier_tail + i] = c_frontier_s[i];
        }
    }
}