#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SomeKernel(int* res, int* data, int col, int row, int y, int step)
{
    // Calculate thread identifier
    unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int currDelta = 0;

    // Iterate columns for each thread
    for (int i = step * threadId; (i < (threadId + 1) * step) && (i < col); i++) {
        // Iterate through rows, from y down to 1
        for (int j = y; j > 0; j--) {
            // Calculate current delta
            currDelta = data[i + j * row] - data[i + (j - 1) * row];

            // Determine magnitude and check threshold
            if (abs(currDelta) > 10) {
                res[i] = j - 1;
                break; // Stop further row processing for this column
            }
        }
    }
}