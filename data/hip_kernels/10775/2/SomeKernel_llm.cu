#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SomeKernel(int* res, int* data, int col, int row, int y, int step)
{
    unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Using shared memory to reduce global memory access
    extern __shared__ int sharedData[];

    for (int i = step * threadId; (i < (threadId + 1) * step) && (i < col); i++) 
    {
        // Load column data into shared memory
        for (int j = 0; j <= y; j++) {
            sharedData[j] = data[i + j * row];
        }
        __syncthreads(); // Ensure all data is loaded before proceeding

        for (int j = y; j > 0; j--) 
        {
            int currDelta = sharedData[j] - sharedData[j - 1];
            // Use absolute difference
            if (abs(currDelta) > 10) {
                res[i] = j - 1;
                break;
            }
        }
    }
}