#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SomeKernel(int* res, int* data, int col, int row, int y, int step)
{
    unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate a per-thread starting column
    int startCol = step * threadId;
    int endCol = min((threadId + 1) * step, col);

    for (int i = startCol; i < endCol; i++) // Iterate over assigned columns
    {
        int lastValue = data[i + y * row]; // Cache last value to avoid redundant loads
        for (int j = y; j > 0; j--) 
        {
            int currValue = data[i + (j - 1) * row];
            int currDelta = lastValue - currValue;
            // Cache to avoid redundant multiplication
            int absDelta = currDelta >= 0 ? currDelta : -currDelta; 
            if (absDelta > 10) 
            {
                res[i] = j - 1;
                break;
            }
            lastValue = currValue; // Update lastValue for next iteration
        }
    }
}