#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

#define delta           10
#define rows            50
#define columns         50

__global__ void SomeKernel(int* res, int* data, int col, int row, int y, int step)
{
    unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by using shared memory for frequently accessed variables
    extern __shared__ int sharedData[];

    // Preload data into shared memory
    if (threadId < col) {
        for (int j = 0; j <= y; j++) {
            sharedData[threadId + j * row] = data[threadId + j * row];
        }
    }
    __syncthreads();

    int currDelta = 0;
    int index = step * threadId;

    // Process columns allocated to this thread
    if (index < col) {
        for (int i = index; i < (threadId + 1) * step && i < col; i++) {
            for (int j = y; j > 0; j--) {
                // Load data from shared memory
                currDelta = sharedData[i + j * row] - sharedData[i + (j - 1) * row];
                
                // Efficient absolute value calculation
                if (abs(currDelta) > delta) {
                    res[i] = j - 1;
                    break;
                }
            }
        }
    }
}