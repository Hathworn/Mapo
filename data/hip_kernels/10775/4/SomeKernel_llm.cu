#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SomeKernel(int* res, int* data, int col, int row, int y, int step)
{
    unsigned int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for better performance when accessing data array
    extern __shared__ int sharedData[];
    
    // Load data into shared memory
    for (int i = threadId * step; i < (threadId + 1) * step && i < col; ++i) {
        sharedData[i] = data[i];
    }
    __syncthreads();
    
    int currDelta = 0;
    for (int i = step * threadId; (i < (threadId + 1) * step) && (i < col); i++) {
        for (int j = y; j > 0; --j) {
            currDelta = sharedData[i + j * row] - sharedData[i + (j - 1) * row];
            // Calculate abs(currDelta) without ternary operator
            if (abs(currDelta) > 10) {
                res[i] = j - 1;
                break;
            }
        }
    }
}
```
