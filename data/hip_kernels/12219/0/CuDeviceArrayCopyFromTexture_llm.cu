#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

texture<float4, 2, hipReadModeElementType> inTex;

// round up n/m
__global__ void CuDeviceArrayCopyFromTexture( float3* dst, int dstStep, int width, int height )
{
    // Calculate the global x and y indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if we are within bounds (>= instead of > for correct boundary checking)
    if ( x >= width || y >= height ) return;

    // Use texture fetching for better memory coalescing
    float4 res = tex2D(inTex, x, y);

    // Calculate the pointer to the specific row and copy data
    float3* row_y = (float3*)((char*)dst + y * dstStep);
    row_y[x] = make_float3(res.x, res.y, res.z);
}