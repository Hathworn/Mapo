#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernBlockWiseSum(const size_t numPoints, const size_t pointDim, double* dest) {
    // Assumes a 2D grid of 1024x1 1D blocks
    int b = blockIdx.y * gridDim.x + blockIdx.x;
    int i = b * blockDim.x + threadIdx.x;

    // Declare shared memory block
    extern __shared__ double blockSum[];

    // Load data into shared memory or set to 0 if out of bounds
    blockSum[threadIdx.x] = (i >= numPoints) ? 0.0 : dest[i * pointDim];
    __syncthreads();

    // Reduce within the block
    for (int s = 1; s < blockDim.x; s *= 2) {
        int index = 2 * s * threadIdx.x;
        if (index < blockDim.x) {
            blockSum[index] += blockSum[index + s];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (threadIdx.x == 0) {
        dest[b * pointDim] = blockSum[0];
    }
}