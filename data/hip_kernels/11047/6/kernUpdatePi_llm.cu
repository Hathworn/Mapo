#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernUpdatePi( const size_t numPoints, const size_t numComponents, double* logpi, double* Gamma ) {
    int b = blockIdx.y * gridDim.x + blockIdx.x;
    int comp = b * blockDim.x + threadIdx.x;
    if(comp >= numComponents) { // Fix boundary condition
        return;
    }

    // Optimize shared memory usage
    __shared__ double A[1024];
    double componentGamma = log(Gamma[comp * numPoints]);
    A[threadIdx.x] = logpi[comp] + componentGamma;
    __syncthreads();

    // Reduce across components using threads
    double sum = 0;
    for(int offset = 0; offset < blockDim.x; ++offset) {
        if(offset < numComponents) {
            sum += exp(A[offset]);
        }
    } 
    
    if(comp < numComponents) { // Ensure writing back only valid components
        logpi[comp] = A[threadIdx.x] - log(sum);
    }
}