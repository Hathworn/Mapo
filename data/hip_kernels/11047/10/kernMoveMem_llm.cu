#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernMoveMem(const size_t numPoints, const size_t pointDim, const size_t s, double* A) {
    int b = blockIdx.y * gridDim.x + blockIdx.x;
    int i = b * blockDim.x + threadIdx.x;

    // Improved use of shared memory with better coalesced access
    __shared__ double mem[1024];

    size_t idx = s * i * pointDim;
    if (idx < numPoints * pointDim) {
        // Load data into shared memory, assuming total threads fit within shared memory size
        mem[threadIdx.x] = A[idx];
    }
    __syncthreads();

    idx = i * pointDim;
    if (idx < numPoints * pointDim) {
        A[idx] = mem[threadIdx.x];
    }
}