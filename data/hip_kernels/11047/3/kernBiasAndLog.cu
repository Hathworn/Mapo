#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernBiasAndLog(double* sumexp, double* bias) {
*sumexp = *bias + log(*sumexp);
}