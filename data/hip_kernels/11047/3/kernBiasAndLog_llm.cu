#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernBiasAndLog(double* sumexp, double* bias) {
    // Load data from global memory to register for faster access
    double local_sumexp = *sumexp;
    double local_bias = *bias;
    
    // Perform bias addition and logarithm operation
    local_sumexp = local_bias + log(local_sumexp);
    
    // Store the result back to global memory
    *sumexp = local_sumexp;
}