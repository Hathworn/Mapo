#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernCalcMu(const size_t numPoints, const size_t pointDim, const double* X, const double* loggamma, const double* GammaK, double* dest) {
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numPoints) {
        return;
    }

    // Cache repeated calculations to reduce redundant computations
    const double gammaExp = exp(loggamma[i]) / exp(*GammaK);
    const double* x = &X[i * pointDim];
    double* y = &dest[i * pointDim];

    // Use shared memory to store pointDim calculations for this thread (optional for large pointDim)
    extern __shared__ double sharedMem[];
    double* sharedX = sharedMem;
    double* sharedY = &sharedMem[pointDim];

    // Load data into shared memory
    for (size_t j = threadIdx.x; j < pointDim; j += blockDim.x) {
        sharedX[j] = x[j];
    }
    __syncthreads();

    // Perform the calculation using shared memory
    for (size_t j = threadIdx.x; j < pointDim; j += blockDim.x) {
        sharedY[j] = gammaExp * sharedX[j];
    }
    __syncthreads();

    // Write results from shared memory back to global memory
    for (size_t j = threadIdx.x; j < pointDim; j += blockDim.x) {
        y[j] = sharedY[j];
    }
}