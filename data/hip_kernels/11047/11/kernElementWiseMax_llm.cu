#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernElementWiseMax(const size_t numPoints, double* dest, double* src) {
    // Calculate flattened global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds and perform max operation
    if (i < numPoints) {
        dest[i] = max(dest[i], src[i]);
    }
}