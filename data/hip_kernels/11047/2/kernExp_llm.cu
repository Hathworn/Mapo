#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernExp(double* A, double* bias) {
    // Cache bias in shared memory for faster access
    __shared__ double shared_bias;
    if (threadIdx.x == 0) {
        shared_bias = *bias;
    }
    __syncthreads();

    int b = blockIdx.y * gridDim.x + blockIdx.x;
    int i = b * blockDim.x + threadIdx.x;
    if (i < gridDim.x * gridDim.y * blockDim.x) {  // Check boundary
        A[i] = exp(A[i] - shared_bias);
    }
}