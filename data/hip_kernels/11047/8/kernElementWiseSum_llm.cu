#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void devVecAdd(size_t pointDim, double* dest, double* src) {
    // Optimize loop with unrolling for better performance
    for(size_t i = 0; i < pointDim; i += 4) {
        if(i < pointDim) dest[i] += src[i];
        if(i + 1 < pointDim) dest[i + 1] += src[i + 1];
        if(i + 2 < pointDim) dest[i + 2] += src[i + 2];
        if(i + 3 < pointDim) dest[i + 3] += src[i + 3];
    }
}

__global__ void kernElementWiseSum(const size_t numPoints, const size_t pointDim, double* dest, double* src) {
    // Calculate global thread ID for 1D grid and 1D block
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure i is within range
    if(i < numPoints) {
        devVecAdd(pointDim, &dest[i * pointDim], &src[i * pointDim]);
    }
}