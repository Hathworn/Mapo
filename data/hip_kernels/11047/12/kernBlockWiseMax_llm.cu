#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernBlockWiseMax(const size_t numPoints, double* dest) {
// Assumes a 2D grid of 1024x1 1D blocks
int b = blockIdx.y * gridDim.x + blockIdx.x;
int i = b * blockDim.x + threadIdx.x;

__shared__ double blockMax[1024];

// Initialize shared memory with a check to avoid out of bounds
blockMax[threadIdx.x] = (i < numPoints) ? dest[i] : -INFINITY;

__syncthreads();

// Use single loop with half sized step optimization
for(int s = blockDim.x / 2; s > 0; s >>= 1) {
    if(threadIdx.x < s && blockMax[threadIdx.x] < blockMax[threadIdx.x + s]) {
        blockMax[threadIdx.x] = blockMax[threadIdx.x + s];
    }
    __syncthreads();
}

if(threadIdx.x == 0) {
// Just do one global write
dest[b] = blockMax[0];
}
}