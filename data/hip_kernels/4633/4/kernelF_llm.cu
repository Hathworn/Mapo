#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelF(const float *d_xAx, const float *d_bx, const float *d_c, float *d_y)
{
    // Simplified calculation using shared variables; assuming operation on single element arrays
    float xAx = *d_xAx;  // Cache the value of d_xAx
    float bx = *d_bx;    // Cache the value of d_bx
    float c = *d_c;      // Cache the value of d_c
    
    *d_y = xAx + bx + c; // Perform the addition with cached values
}