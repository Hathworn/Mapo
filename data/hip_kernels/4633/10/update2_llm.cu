#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update2(float *alphaMinusBeta_out, const float *rho, const float *yDotZ, const float *alpha)
{
    // Use shared memory to minimize global memory access latency
    __shared__ float shared_rho;
    __shared__ float shared_yDotZ;
    __shared__ float shared_alpha;

    // Load the variables into shared memory
    if (threadIdx.x == 0) {
        shared_rho = *rho;
        shared_yDotZ = *yDotZ;
        shared_alpha = *alpha;
    }
    __syncthreads();

    // Compute beta using shared memory
    const float beta = shared_rho * shared_yDotZ;

    // Calculate alphaMinusBeta using shared memory
    *alphaMinusBeta_out = shared_alpha - beta;
}