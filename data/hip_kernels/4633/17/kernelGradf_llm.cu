#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelGradf(const float *d_x, float *d_grad)
{
    // Load inputs into registers to avoid multiple global memory accesses
    const float x0 = d_x[0];
    const float x1 = d_x[1];

    // Compute reusable expression - (x1 - x0^2) to reduce redundancy
    const float x1_minus_x0_squared = x1 - x0 * x0;

    // Use reusable expression to calculate gradients
    d_grad[0] = -2.0f * (1.0f - x0) - 400.0f * x0 * x1_minus_x0_squared;
    d_grad[1] = 200.0f * x1_minus_x0_squared;
}
```
