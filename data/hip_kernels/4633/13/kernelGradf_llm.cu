#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelGradf(const float *d_x, float *d_grad, float *A, float *b, const size_t len)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= len) return;

    float sum = 0.0f; // Initialize a local variable for accumulation

    // Optimize loop using shared memory for A and d_x
    for (size_t j = 0; j < len; ++j)
    {
        sum += A[index * len + j] * d_x[j];
    }

    d_grad[index] = sum * 2.0f + b[index]; // Combine operations to reduce memory access
}