#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelF(const float *d_xAx, const float *d_bx, const float *d_c, float *d_y)
{
    // Utilize threadIdx.x for potential concurrent execution
    int idx = threadIdx.x;
    // Calculate result in a more concurrent-friendly manner
    d_y[idx] = d_xAx[idx] + d_bx[idx] + d_c[idx];
}