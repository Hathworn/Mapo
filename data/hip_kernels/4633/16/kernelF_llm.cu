#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelF(const float *d_x, float *d_y)
{
    // Load input data into registers to optimize memory access
    float x0 = d_x[0];
    float x1 = d_x[1];

    // Perform calculations using registers
    float a = (1.0f - x0);
    float b = (x1 - x0 * x0);

    // Store the result
    *d_y = a * a + 100.0f * b * b;
}