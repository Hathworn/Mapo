#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update3(float *rho_out, float *H0_out, const float *yDotS, const float *yDotY)
{
    // Use local variables to prevent repeated dereferencing
    float yDotS_value = *yDotS;
    float yDotY_value = *yDotY;

    // Perform division in one access
    *rho_out = 1.0f / yDotS_value;

    // Conditional check with local variable
    if (yDotY_value > 1e-5)
        *H0_out = yDotS_value / yDotY_value;
}