#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelGradf(const float *d_x, float *d_grad, float *A, float *b, const size_t len)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if index is out of bounds
    if (index >= len)
        return;

    float sum = 0.0f;

    // Use shared memory for improved data locality
    extern __shared__ float shared_x[];
    if (threadIdx.x < len)
    {
        shared_x[threadIdx.x] = d_x[threadIdx.x];
    }
    __syncthreads();

    // Accumulate the dot product
    for (size_t j = 0; j < len; ++j)
    {
        sum += A[index * len + j] * shared_x[j];
    }

    // Perform final gradient calculation
    d_grad[index] = 2.0f * sum + b[index];
}