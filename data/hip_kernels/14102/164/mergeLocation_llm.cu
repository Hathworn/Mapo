#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized mergeLocation kernel function
__global__ void mergeLocation(const short2* __restrict__ loc_, float* __restrict__ x, float* __restrict__ y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid unnecessary branching and memory access within bounds
    if (ptidx >= npoints) return;
    
    short2 loc = loc_[ptidx];
    
    // Direct computation to reduce instruction count
    x[ptidx] = __fmul_rn(loc.x, scale);
    y[ptidx] = __fmul_rn(loc.y, scale);
}