#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image) {
    // Calculate unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate the linear position
    int pos = idy * w + idx;

    // Check if within bounds
    if (idx < w && idy < h) {
        image[pos] = value;
    }
}