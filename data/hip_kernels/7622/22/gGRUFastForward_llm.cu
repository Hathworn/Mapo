#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableLogit(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gGRUFastForward(float* out, const float* state, const float* xW, const float* sU, const float* b, const float* mask, size_t rows, size_t cols, bool final) {
    int j = blockIdx.x + blockIdx.y * gridDim.x;
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if(j < rows && i < cols) {
        float m = !mask || mask[j];
        float* rowOut = out + j * cols;
        const float* rowState = state + j * cols;

        const float* xWrow = xW + j * cols * 3;
        const float* sUrow = sU + j * cols * 3;

        float r = stableLogit(xWrow[i] + sUrow[i] + b[i]);
        int k = i + cols;
        float z = stableLogit(xWrow[k] + sUrow[k] + b[k]);
        int l = i + 2 * cols;

        float h = final ? tanhf(xWrow[l] + (sUrow[l] + b[l]) * r) : tanhf(xWrow[l] + sUrow[l] * r + b[l]);
        float curOut = (1.0f - z) * h + z * rowState[i];
        rowOut[i] = m * curOut + (1 - m) * rowState[i];
    }
}