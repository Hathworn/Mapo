#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableLogit(float x) {
    if (x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gLSTMCellBackward(float* outCell, float* outXW, float* outSU, float* outB, const float* cell, const float* xW, const float* sU, const float* b, const float* mask, const float* adj, size_t rows, size_t cols) {
    int j = blockIdx.x + blockIdx.y * gridDim.x; // Utilize 2D grid for better scalability
    if (j < rows) {
        int i = threadIdx.x;
        
        float m = !mask || mask[j];
        
        float* rowOutCell = outCell + j * cols;
        float* rowOutXW = outXW + j * cols * 4;
        float* rowOutSU = outSU + j * cols * 4;
        
        const float* rowCell = cell + j * cols;
        const float* xWrow = xW + j * cols * 4;
        const float* sUrow = sU + j * cols * 4;
        
        const float* rowAdj = adj + j * cols;

        while (i < cols) { // Utilize while loop for iteration
            float gf = stableLogit(xWrow[i] + sUrow[i] + b[i]);
            int k = i + cols;
            float gi = stableLogit(xWrow[k] + sUrow[k] + b[k]);
            int l = i + 2 * cols;
            float gc = tanhf(xWrow[l] + sUrow[l] + b[l]);
            
            float adjValue = rowAdj[i];

            if (outCell)
                rowOutCell[i] += (m * gf - m + 1) * adjValue;

            float dcdxf = m * rowCell[i] * gf * (1 - gf) * adjValue;
            if (outXW)
                rowOutXW[i] += dcdxf;
            if (outSU)
                rowOutSU[i] += dcdxf;
            if (outB)
                atomicAdd(outB + i, dcdxf);

            float dcdb_i = m * gc * gi * (1 - gi) * adjValue;
            if (outXW)
                rowOutXW[k] += dcdb_i;
            if (outSU)
                rowOutSU[k] += dcdb_i;
            if (outB)
                atomicAdd(outB + k, dcdb_i);

            float dcdxc = m * gi * (1 - gc * gc) * adjValue;
            if (outXW)
                rowOutXW[l] += dcdxc;
            if (outSU)
                rowOutSU[l] += dcdxc;
            if (outB)
                atomicAdd(outB + l, dcdxc);

            i += blockDim.x; // Thread-level parallelism
        }
    }
}