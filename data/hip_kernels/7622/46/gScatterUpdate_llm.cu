#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gScatterUpdate(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    // Calculate global index once
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Early exit for out-of-bounds threads
    if(idx >= sparseSize) return;
    
    // Store index calculation in a variable to avoid redundant computation
    int targetIndex = sparseIndices[idx] + offset;
    
    // Check boundaries
    if(sparseIndices[idx] >= -offset && targetIndex < denseSize) {
        // Perform scatter update
        denseData[targetIndex] = sparseData[idx];
    }
}