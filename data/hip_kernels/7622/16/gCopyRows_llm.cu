#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gCopyRows(float* out, const float* in, size_t cols, const size_t* sourceRowIdx, size_t rows) {
    // Use a single block per row and a single thread per column element for better utilization
    int j = blockIdx.x;
    if(j < rows) {
        size_t dstId = j;
        size_t srcId = sourceRowIdx[j];
        
        float* rowOut = out + dstId * cols;
        const float* rowIn = in + srcId * cols;
        
        int i = threadIdx.x;
        if(i < cols)
            rowOut[i] = rowIn[i];
    }
}