#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gMaxPoolingForward(float* out, int outRows, int outCols, const float* in, int inRows, int inCols, const float* mask, int numKernels, int maskCols, int width, int lastWidth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid >= outRows * outCols)
        return;

    // Compute row and column indices
    int rowId = tid / outCols;  // Fix division by outCols
    int colId = tid % outCols;

    const float* b = in + (rowId * inCols) + (colId * width);
    const float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;

    // Adjust width for the last column
    int effectiveWidth = (colId == outCols - 1) ? lastWidth : width;

    // Initialize with first element's product
    float currentMax = b[0] * localMask[0];
    
    // Unroll loop to reduce iteration overhead
    #pragma unroll
    for(int i = 1; i < effectiveWidth; ++i) {
        float candidate = b[i] * localMask[i];
        if(candidate > currentMax) {
            currentMax = candidate;
        }
    }
    
    out[rowId + (colId * outCols)] = currentMax;
}