#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableLogit(float x) {
  if (x >= 0) {
    float z = expf(-x);
    return 1.0 / (1.0 + z);
  } else {
    float z = expf(x);
    return z / (1.0 + z);
  }
}

__global__ void gHighwayBackward(
  float* out1, 
  float* out2, 
  float* outt, 
  const float* in1, 
  const float* in2, 
  const float* t, 
  const float* adj, 
  size_t length) {
    
  int index = blockDim.x * blockIdx.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  // Use strided loop for memory coalescing and better occupancy
  for (int i = index; i < length; i += stride) {
    float sigma = stableLogit(t[i]);
    float adjVal = adj[i];
    out1[i] = sigma * adjVal;
    out2[i] = (1.f - sigma) * adjVal; 
    outt[i] = sigma * (1.f - sigma) * (in1[i] - in2[i]) * adjVal;
  }
}