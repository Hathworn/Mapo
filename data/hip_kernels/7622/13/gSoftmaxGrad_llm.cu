#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gSoftmaxGrad(float* grad, const float* adj, const float* val, const int rows, const int cols) {
    // Calculate row based on grid and block index
    int j = blockIdx.x + blockIdx.y * gridDim.x;
    if(j < rows) {
        extern __shared__ float _share[];
        float* _sum = _share;

        float* gradRow = grad + j * cols;
        const float* adjRow = adj + j * cols;
        const float* valRow = val + j * cols;

        // Initialize shared memory for summing
        float tempSum = 0.0f;
        for(int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
            tempSum += valRow[tid] * adjRow[tid];
        }

        _sum[threadIdx.x] = tempSum;
        __syncthreads();

        // Reduce sum in shared memory
        for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
            if(threadIdx.x < offset) {
                _sum[threadIdx.x] += _sum[threadIdx.x + offset];
            }
            __syncthreads();
        }

        // Calculate gradients
        float finalSum = _sum[0];
        for(int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
            float val = valRow[tid] * (adjRow[tid] - finalSum);
            if(val) {
                gradRow[tid] += val;
            }
        }
    }
}