#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gJoin2(float* out, size_t rowBatch, size_t cols, const float* in1, size_t inStride1, const float* in2, size_t inStride2) {
    int outStride = inStride1 + inStride2;
    int rows = rowBatch * outStride;
    int j = blockIdx.x + blockIdx.y * gridDim.x; // Utilize 2D grid

    if (j < rows) {
        float* rowOut = out + j * cols;
        int curBatch = j / outStride;
        int curPos = j % outStride;

        int jIn1 = (curBatch * inStride1) + curPos;
        int jIn2 = (curBatch * inStride2) + curPos - inStride1;

        const float* rowIn1 = in1 + jIn1 * cols;
        const float* rowIn2 = in2 + jIn2 * cols;

        // Use thread-level parallelism to load rows more efficiently
        int i = threadIdx.x;
        if (i < cols) {
            rowOut[i] = (curPos < inStride1) ? rowIn1[i] : rowIn2[i];
        }
    }
}