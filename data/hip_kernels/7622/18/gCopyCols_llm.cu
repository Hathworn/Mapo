#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gCopyCols(float* out, const float* in, size_t rows, size_t colsIn, const size_t* sourceColIdx, size_t colsOut) {
    int j = blockIdx.x + blockIdx.y * gridDim.x; // Determine row index using 2D block grid
    if (j < rows) {
        const float* rowIn = in + j * colsIn;
        float* rowOut = out + j * colsOut;

        for (int i = threadIdx.x; i < colsOut; i += blockDim.x) { // Improved parallel access within the block
            if (i < colsOut)
                rowOut[i] = rowIn[sourceColIdx[i]];
        }
    }
}