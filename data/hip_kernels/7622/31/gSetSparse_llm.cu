#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gSetSparse(float* out, const size_t* indices, const float* values, int length) {
    // Optimize: use a single loop with stride for better parallelization
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < length; i += stride) {
        out[indices[i]] = values[i];
    }
}