#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gScatterAdd(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if idx is within the bounds to avoid unnecessary threads
    if(idx < sparseSize) {
        int index = sparseIndices[idx] + offset;
        
        // Check the calculated index once for bounds
        if(index >= 0 && index < denseSize) {
            atomicAdd(&denseData[index], sparseData[idx]); // Ensure safe concurrent writes
        }
    }
}