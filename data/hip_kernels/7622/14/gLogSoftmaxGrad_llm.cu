#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gLogSoftmaxGrad(float* grad, const float* adj, const float* val, const int rows, const int cols) {
    // Using a single block to process multiple rows
    for (int bid = blockIdx.x; bid < rows; bid += gridDim.x) {
        int j = bid;
        if (j < rows) {
            extern __shared__ float _share[];
            float* _sum = _share;

            float* gradRow = grad + j * cols;
            const float* adjRow = adj + j * cols;
            const float* valRow = val + j * cols;

            // Reset _sum for current row
            _sum[threadIdx.x] = 0.0;
            for (int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
                // Calculate partial sum using each thread
                _sum[threadIdx.x] += adjRow[tid];
            }
            __syncthreads();

            // Reduce partial sums to compute the total sum
            for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
                if (threadIdx.x < offset) {
                    _sum[threadIdx.x] += _sum[threadIdx.x + offset];
                }
                __syncthreads();
            }

            // Calculate gradient adjustments using total sum
            for (int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
                gradRow[tid] += adjRow[tid] - (expf(valRow[tid]) * _sum[0]);
            }
        }
    }
}