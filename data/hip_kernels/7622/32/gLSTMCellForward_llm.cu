#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableLogit(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gLSTMCellForward(float* out, const float* cell, const float* xW, const float* sU, const float* b, const float* mask, size_t rows, size_t cols) {
    // Use 2D grid to handle rows and columns simultaneously
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols) {
        float m = !mask || mask[row];

        float* rowOut = out + row * cols;
        const float* rowCell = cell + row * cols;

        const float* xWrow = xW + row * cols * 4;
        const float* sUrow = sU + row * cols * 4;

        // Fetch weights and biases preemptively
        float b_i = b[col];
        float xWrow_i = xWrow[col];
        float sUrow_i = sUrow[col];

        float gf = stableLogit(xWrow_i + sUrow_i + b_i);

        int k = col + cols;
        float gi = stableLogit(xWrow[k] + sUrow[k] + b[k]);

        int l = col + 2 * cols;
        float gc = tanhf(xWrow[l] + sUrow[l] + b[l]);

        float cout = gf * rowCell[col] + gi * gc;
        rowOut[col] = m * cout + (1 - m) * rowCell[col];
    }
}