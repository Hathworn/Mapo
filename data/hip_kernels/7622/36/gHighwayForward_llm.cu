#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableLogit(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gHighwayForward(float* out, const float* in1, const float* in2, const float* t, size_t length) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;  // Calculate global thread index
    int stride = blockDim.x * gridDim.x;                // Calculate total number of active threads

    for(int i = index; i < length; i += stride) {       // Stride loop to handle large data
        float sigma = stableLogit(t[i]);
        out[i] = in1[i] * sigma + in2[i] * (1.f - sigma);
    }
}