#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableLogit(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0f / (1.0f + z);
    } else {
        float z = expf(x);
        return z / (1.0f + z);
    }
}

__global__ void gLSTMOutputForward(float* out, const float* cell, const float* xW, const float* sU, const float* b, size_t rows, size_t cols) {
    int j = blockIdx.x + blockIdx.y * gridDim.x;  // Compute global row index
    if(j < rows) {
        float* rowOut = out + j * cols;
        const float* rowCell = cell + j * cols;
        const float* xWrow = xW + j * cols * 4;
        const float* sUrow = sU + j * cols * 4;

        int i = threadIdx.x + blockIdx.y * blockDim.x;  // Compute global column index
        while(i < cols) {  // Use a while loop to cover all elements along the row
            int k = i + 3 * cols;
            float go = stableLogit(xWrow[k] + sUrow[k] + b[k]);
            rowOut[i] = go * tanhf(rowCell[i]);
            i += blockDim.x * gridDim.x;  // Move to the next column index
        }
    }
}