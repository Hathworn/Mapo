#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gMaxPoolingBackward(float* adj, int adjRows, int adjCols, float* in, float* adjIn, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Exit early if thread index exceeds total output elements
    if(tid >= adjRows * adjCols) return;

    int rowId = tid / adjRows;
    int colId = tid % adjRows;

    // Calculate input start position for the current thread's output element
    float* b = in + (rowId * inCols) + (colId * width);

    // Adjust width for the last partition
    if(colId == adjRows - 1) {
        width = lastWidth;
    }

    // Calculate localMask start position
    float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;
    size_t currentMaxIdx = 0;

    // Find the index of the maximum value multiplied by mask in the current window
    for(int i = 1; i < width; ++i) {
        if(b[i] * localMask[i] > b[currentMaxIdx] * localMask[currentMaxIdx]) {
            currentMaxIdx = i;
        }
    }

    // Safely update the adjIn array using atomic operation
    atomicAdd(&adjIn[(rowId * inCols) + (colId * width) + currentMaxIdx],
              adj[rowId + (colId * adjCols)]);
}