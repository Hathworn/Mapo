#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gPasteCols(float* out, const float* in, size_t rows, size_t colsOut, const size_t* targetColIdx, size_t colsIn) {
    // Calculate global thread index for row processing
    int j = blockIdx.x * blockDim.y + threadIdx.y;
    if (j < rows) {
        const float* rowIn = in + j * colsIn;
        float* rowOut = out + j * colsOut;
        
        // Calculate global thread index for column processing
        for (int i = threadIdx.x; i < colsIn; i += blockDim.x) {
            rowOut[targetColIdx[i]] = rowIn[i]; // Map input columns to output using target indices
        }
    }
}