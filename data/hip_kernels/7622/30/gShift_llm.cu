#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gShift(float* out, const float* in, int length, int offset) {
    // Calculate the global index for the current thread.
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Use single loop iteration for simplicity and efficiency.
    if (index < length) {
        // Check index boundaries to prevent memory access violations.
        if (index - offset < 0 || index - offset >= length)
            out[index] = 0;
        else
            out[index] = in[index - offset];
    }
}