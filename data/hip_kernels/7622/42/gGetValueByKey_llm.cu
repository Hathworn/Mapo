#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gGetValueByKey(float* d_in, float* d_out, int* indeces, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Order blockIdx and blockDim calculations
    if (tid < n) {
        int index = __ldg(&indeces[tid]); // Use __ldg for read-only memory access
        d_out[tid] = __ldg(&d_in[index]); // Use __ldg for read-only memory access
    }
}