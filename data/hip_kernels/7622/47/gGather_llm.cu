#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gGather(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(idx < sparseSize) {
        int adjustedIndex = sparseIndices[idx] + offset;
        // Ensure the adjusted index is within bounds
        if(adjustedIndex >= 0 && adjustedIndex < denseSize) {
            sparseData[idx] = denseData[adjustedIndex];
        }
    }
}