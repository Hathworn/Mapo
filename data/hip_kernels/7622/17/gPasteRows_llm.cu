#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gPasteRows(float* out, const float* in, size_t cols, const size_t* targetRowIdx, size_t rows) {
    // Optimize grid stride for better use of block and threads
    for (int j = blockIdx.x; j < rows; j += gridDim.x) {
        size_t dstId = targetRowIdx[j];
        size_t srcId = j;

        float* rowOut = out + dstId * cols;
        const float* rowIn = in + srcId * cols;

        // Optimize block stride for better parallelism
        for (int i = threadIdx.x; i < cols; i += blockDim.x) {
            atomicAdd(rowOut + i, rowIn[i]);
        }
    }
}