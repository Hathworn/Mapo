#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define _CRT_SECURE_NO_WARNINGS

#define BLOCK_DIM 16

__global__ void transposition(int* matrix, int* matrixOut, int length, int width)
{
    __shared__ int tempMatrix[BLOCK_DIM][BLOCK_DIM + 1]; // Avoid bank conflicts

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Load data into shared memory if within bounds
    if (i < length && j < width)
    {
        tempMatrix[threadIdx.y][threadIdx.x] = matrix[j * length + i];
    }

    __syncthreads();

    // Transpose the block and write to output if within bounds
    i = blockIdx.y * blockDim.y + threadIdx.x;
    j = blockIdx.x * blockDim.x + threadIdx.y;

    if (i < width && j < length)
    {
        matrixOut[j * width + i] = tempMatrix[threadIdx.x][threadIdx.y];
    }
}