#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {


}

__global__ void vmul(const float *a, const float *b, float *c, int n)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (i < n) {
        c[i] = a[i] * b[i];
    }
}