#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

}

// Kernel function to perform vector division with boundary check
__global__ void vdiv(const float *a, const float *b, float *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) { // Ensure within bounds
        c[i] = a[i] / b[i];
    }
}