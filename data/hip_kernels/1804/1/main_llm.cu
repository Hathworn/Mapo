#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "mandel_double_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    uchar4 *ptr = NULL;
hipMalloc(&ptr, XSIZE*YSIZE);
int max_w = 1;
int max_h = 1;
int w = XSIZE;
int h = YSIZE;
double cx = 1;
double cy = 1;
double zoom = 1;
int iter_mult = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
mandel_double<<<gridBlock, threadBlock>>>(ptr,max_w,max_h,w,h,cx,cy,zoom,iter_mult);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
mandel_double<<<gridBlock, threadBlock>>>(ptr,max_w,max_h,w,h,cx,cy,zoom,iter_mult);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
mandel_double<<<gridBlock, threadBlock>>>(ptr,max_w,max_h,w,h,cx,cy,zoom,iter_mult);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}