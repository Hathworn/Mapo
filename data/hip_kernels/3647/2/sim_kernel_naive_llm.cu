#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define IDX2D(a, i, stride, j) ((a)[(i)*(stride) + (j)])

__global__ void sim_kernel_naive(double *z, double *v, size_t nx, size_t ny, double dx2inv, double dy2inv, double dt) {
    const int mesh_x = blockIdx.x * blockDim.x + threadIdx.x + 1;
    const int mesh_y = blockIdx.y * blockDim.y + threadIdx.y + 1;

    // Check if indices are within bounds
    if (mesh_x >= nx - 1 || mesh_y >= ny - 1) return;

    // Caching values in registers for faster access
    const double z_left = IDX2D(z, mesh_y, nx, mesh_x - 1);
    const double z_right = IDX2D(z, mesh_y, nx, mesh_x + 1);
    const double z_top = IDX2D(z, mesh_y - 1, nx, mesh_x);
    const double z_bottom = IDX2D(z, mesh_y + 1, nx, mesh_x);
    const double z_val = IDX2D(z, mesh_y, nx, mesh_x);

    const double ax = dx2inv * (z_left + z_right - 2.0 * z_val);
    const double ay = dy2inv * (z_top + z_bottom - 2.0 * z_val);

    // Store results directly to global memory
    double v_val = IDX2D(v, mesh_y, nx, mesh_x) + dt * (ax + ay) / 2.0;
    IDX2D(v, mesh_y, nx, mesh_x) = v_val;
    IDX2D(z, mesh_y, nx, mesh_x) += dt * v_val;
}