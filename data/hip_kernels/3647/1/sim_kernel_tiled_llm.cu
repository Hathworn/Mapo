#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define IDX2D(a, i, stride, j) ((a)[(i)*(stride) + (j)])

__global__ void sim_kernel_tiled(double *z, double *v, size_t nx, size_t ny, double dx2inv, double dy2inv, double dt) {
    extern __shared__ double z_tile[];

    const int block_mesh_x = blockDim.x * blockIdx.x + 1;
    const int block_mesh_y = blockDim.y * blockIdx.y + 1;

    const int mesh_xx = block_mesh_x + threadIdx.x;
    const int mesh_xy = block_mesh_y + threadIdx.y;

    // Load z values into shared memory
    if (mesh_xx < nx && mesh_xy < ny)
        z_tile[threadIdx.y * blockDim.x + threadIdx.x] = IDX2D(z, mesh_xy, nx, mesh_xx);
    __syncthreads();

    if (mesh_xx >= nx-1 || mesh_xy >= ny-1)
        return;

    double ax, ay;
    // Compute ax using shared memory when possible
    ax = dx2inv * (
        (threadIdx.x > 0 ? IDX2D(z_tile, threadIdx.y, blockDim.x, threadIdx.x - 1) : IDX2D(z, mesh_xy, nx, mesh_xx - 1)) +
        (threadIdx.x < blockDim.x - 1 ? IDX2D(z_tile, threadIdx.y, blockDim.x, threadIdx.x + 1) : IDX2D(z, mesh_xy, nx, mesh_xx + 1)) -
        2.0 * z_tile[threadIdx.y * blockDim.x + threadIdx.x]
    );

    // Compute ay using shared memory when possible
    ay = dy2inv * (
        (threadIdx.y > 0 ? IDX2D(z_tile, threadIdx.y - 1, blockDim.x, threadIdx.x) : IDX2D(z, mesh_xy - 1, nx, mesh_xx)) +
        (threadIdx.y < blockDim.y - 1 ? IDX2D(z_tile, threadIdx.y + 1, blockDim.x, threadIdx.x) : IDX2D(z, mesh_xy + 1, nx, mesh_xx)) -
        2.0 * z_tile[threadIdx.y * blockDim.x + threadIdx.x]
    );

    // Update v and z values
    const double v_val = IDX2D(v, mesh_xy, nx, mesh_xx) += (ax + ay) / 2.0 * dt;
    IDX2D(z, mesh_xy, nx, mesh_xx) += dt * v_val;
}