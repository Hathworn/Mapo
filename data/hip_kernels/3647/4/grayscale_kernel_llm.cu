#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define IDX2D(a, i, stride, j) ((a)[(i)*(stride) + (j)])

__global__ void grayscale_kernel(double *z, unsigned char *output, size_t size, double z_min, double z_max) {
    // Calculate global thread index
    const int idx = threadIdx.x + blockDim.x * blockIdx.x;
    const int stride = blockDim.x * gridDim.x;

    // Use shared memory to optimize repeated calculations
    __shared__ double scaled[256]; // Assuming max blockDim.x value
    
    if (threadIdx.x < 256) {
        // Pre-compute the scaling factors
        scaled[threadIdx.x] = z_min + ((z_max - z_min) * threadIdx.x / 255.0);
    }
    __syncthreads();

    // Efficient iteration over the assigned data
    for (int i = idx; i < size; i += stride) {
        double value = z[i];
        int scaled_value = static_cast<int>((value - z_min) / (z_max - z_min) * 255);
        output[i] = (unsigned char) (fmin(fmax(scaled_value, 0), 255)); // Clamp to 0-255
    }
}