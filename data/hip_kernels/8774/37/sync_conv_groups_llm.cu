#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups() { 
    // Synchronize all threads within a block for proper execution order
    __syncthreads(); 
}