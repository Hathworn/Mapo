#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use more blocks and threads for better parallelism
__global__ void chol_kernel_optimized_div(float * U, int k, int stride) {
    // General thread id
    int tx = blockIdx.x * blockDim.x + threadIdx.x;

    // Number of rows in the matrix
    unsigned int num_rows = MATRIX_SIZE;

    // Let only one thread compute the square root of the diagonal element
    if (tx == 0) {
        U[k * num_rows + k] = sqrtf(U[k * num_rows + k]); // Use sqrtf for better performance
    }

    // Calculate the starting point and step size for each thread
    int offset = (k + 1); // Offset for original loop
    int jstart = blockIdx.x * blockDim.x * stride + threadIdx.x * stride + offset;
    int jstep = gridDim.x * blockDim.x * stride; // Ensure all blocks work together

    // Upper and lower bounds on j for the loop
    int jtop = num_rows - 1;
    int jbottom = (k + 1);

    // Let all threads from all blocks perform division step
    for (unsigned int j = jstart; j <= jtop; j += jstep) {
        if (j >= jbottom) {
            U[k * num_rows + j] /= U[k * num_rows + k]; // Division step
        }
    }
}