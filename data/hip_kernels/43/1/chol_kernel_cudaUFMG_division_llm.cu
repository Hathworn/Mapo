#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void chol_kernel_cudaUFMG_division(float * U, int elem_per_thr) {
    // Get a thread identifier
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    int tn = ty * blockDim.x * gridDim.x + tx;
    
    #ifdef DEBUGDIV
    int dbg = 0;
    if(blockIdx.x == 4 && blockIdx.y == 5 && threadIdx.x == 2 && threadIdx.y == 1){
        dbg = 1;
        printf("\n\n");
        printf("\ntx=%d \nty=%d \ntn=%d", tx, ty, tn);
    }
    #endif
    
    for(unsigned i = 0; i < elem_per_thr; i++) {
        int iel = tn * elem_per_thr + i;
        int xval = iel % MATRIX_SIZE;
        int yval = iel / MATRIX_SIZE;
        
        if(xval == yval) {
            continue; // Skip diagonal elements
        }
        
        #ifdef DEBUGDIV
        if(dbg == 1 && i == 37){
            printf("\niel=%d \nxval=%d \nyval=%d", iel, xval, yval);
        }
        #endif
        
        // If on the lower diagonal, adjust indices
        if(yval > xval) {
            xval = MATRIX_SIZE - xval - 1;
            yval = MATRIX_SIZE - yval - 1;
        }
        
        int iU = xval + yval * MATRIX_SIZE;
        int iDiag = yval + yval * MATRIX_SIZE;
        
        #ifdef DEBUGDIV
        if(dbg == 1 && i == 37){
            printf("\nxtrans=%d \nytrans=%d \niU=%d \niDiag=%d", xval, yval, iU, iDiag);
            printf("\n\n");
        }
        #endif
        
        U[iU] /= U[iDiag]; // Division operation optimized
    }
}