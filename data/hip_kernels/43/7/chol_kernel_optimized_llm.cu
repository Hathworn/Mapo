#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void chol_kernel_optimized(float * U, int k, int stride) {
    // Define the number of rows as a constant to avoid recalculating
    const unsigned int num_rows = MATRIX_SIZE;
    
    // Calculate the iteration index for i using block index
    const int i = blockIdx.x + (k + 1);
    
    // Calculate initial j index for this thread and stride step
    const int jstart = threadIdx.x + i;
    const int jstep = stride;
    
    // Limit check bounds for j
    const int jtop = num_rows - 1;
    const int jbottom = i;

    // Using register variables for frequently accessed U elements
    float Uk_i = U[k * num_rows + i];
    
    // Loop with stride for j
    for (int j = jstart; (j >= jbottom) && (j <= jtop); j += jstep) {
        U[i * num_rows + j] -= Uk_i * U[k * num_rows + j];
    }
}