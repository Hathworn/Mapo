#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void chol_kernel_optimized_no_stride(float * U, int k, int stride) {
    // Iterators
    unsigned int num_rows = MATRIX_SIZE;

    // Calculate the current i iteration
    int i = blockIdx.x + (k + 1);

    // Determine the start and end indices for j based on stride and thread index
    int offset = i;
    int jstart = (threadIdx.x * stride) + offset;
    int jend = jstart + (stride - 1);

    // Boundary check to ensure j remains in bounds
    if (jstart <= num_rows - 1) {
        jend = min(jend, num_rows - 1);

        // Perform computations for this i iteration
        for (int j = max(jstart, i); j <= jend; j++) {
            U[i * num_rows + j] -= U[k * num_rows + i] * U[k * num_rows + j];
        }
    }
}