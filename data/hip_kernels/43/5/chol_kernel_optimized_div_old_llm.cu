#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void chol_kernel_optimized_div_old(float * U, int k, int stride) {
    // General thread id
    int tx = blockIdx.x * blockDim.x + threadIdx.x;

    // Iterators
    unsigned int j;
    unsigned int num_rows = MATRIX_SIZE;

    // Only let one thread do this
    if (tx == 0) {
        // Take the square root of the diagonal element
        U[k * num_rows + k] = sqrtf(U[k * num_rows + k]); // Use sqrtf for better performance

        // Unroll loop for Division step
        #pragma unroll
        for (j = (k + 1); j < num_rows; j++) {
            U[k * num_rows + j] /= U[k * num_rows + k]; // Division step
        }
    }
}