#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void chol_kernel(float * U, int ops_per_thread) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int num_rows = MATRIX_SIZE;

    for (unsigned int k = 0; k < num_rows; k++) {
        if (tx == 0) {
            // Compute square root and division on the diagonal
            U[k * num_rows + k] = sqrt(U[k * num_rows + k]);
            for (unsigned int j = k + 1; j < num_rows; j++) {
                U[k * num_rows + j] /= U[k * num_rows + k];
            }
        }
        __syncthreads();

        // Calculate this thread's range of work
        int istart = (k + 1) + tx * ops_per_thread;
        int iend = min(istart + ops_per_thread, (int)num_rows);

        for (unsigned int i = istart; i < iend; i++) {
            for (unsigned int j = i; j < num_rows; j++) {
                U[i * num_rows + j] -= U[k * num_rows + i] * U[k * num_rows + j];
            }
        }
        __syncthreads();
    }

    // Zero out the lower triangular portion of U
    int istart = tx * ops_per_thread;
    int iend = min(istart + ops_per_thread, (int)num_rows);

    for (unsigned int i = istart; i < iend; i++) {
        for (unsigned int j = 0; j < i; j++) {
            U[i * num_rows + j] = 0.0f;
        }
    }
}