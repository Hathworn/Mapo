#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void chol_kernel_cudaUFMG_elimination(float * U, int k) {
    // Calculate the current row based on block index
    int i = (k + 1) + blockIdx.x;

    // Calculate starting index for this thread
    int jstart = i + threadIdx.x;
    int jstep = blockDim.x;

    // Pre-calculate indexes for efficient access
    int kM = k * MATRIX_SIZE;
    int iM = i * MATRIX_SIZE;
    int ki = kM + i;

    float uki = U[ki];  // Load U[ki] to a register for faster repeated access

    // Process the rows, striding according to total threads in the block
    for (int j = jstart; j < MATRIX_SIZE; j += jstep) {
        U[iM + j] -= uki * U[kM + j];  // Perform the row elimination update
    }
}