#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Total number of threads
    const int numThreads = blockDim.x * gridDim.x;

    // Remove division and modulus operations from inside the loop
    const uint numEls = tgtWidth * tgtHeight;
    for (uint i = idx; i < numEls; i += numThreads) {
        const uint y = __umul24(i, 1 / tgtWidth); // Use __umul24 for fast integer division
        const uint x = i - y * tgtWidth; // Replace modulus with subtraction
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}