#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    
    // Shared memory for reduction
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0;

    // Efficient reduction using warp intrinsics
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();
    
    // Perform reduction using the first warp
    if (idx < warpSize) { // Avoid launching unnecessary threads
        tmp = 0;
        for (int i = idx; i < SOFTMAX_NUM_THREADS; i += warpSize) {
            tmp += reduction_buffer[i];
        }
        reduction_buffer[idx] = tmp;
    }
    __syncthreads();

    // Use a single thread to aggregate results
    if (idx == 0) {
        tmp = 0;
        for (int i = 0; i < warpSize; ++i) {
            tmp += reduction_buffer[i];
        }
        reduction_buffer[0] = tmp;
    }
    __syncthreads();
    
    // Compute gradient
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}