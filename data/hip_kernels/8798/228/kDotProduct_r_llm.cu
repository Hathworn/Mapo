#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;

    // Load elements and compute partial dot products
    if (eidx < numElements) {
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            shmem[threadIdx.x] += a[eidx] * b[eidx];
        }
    }
    __syncthreads();

    // Unroll the loop for better performance
    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    __syncthreads();
    
    // Reduce using warp shuffle for the last 64 elements
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem; // volatile to ensure correctness of warp shuffle
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    // Write result to target
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }    
}