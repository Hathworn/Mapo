#include "hip/hip_runtime.h"
```c++
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling4(int *g_idata, int *g_odata, unsigned int n) {
    // Cache thread and block info
    unsigned int tid = threadIdx.x;
    unsigned int idx = (4 * blockIdx.x) * blockDim.x + tid;
    
    int *idata = g_idata + (4 * blockIdx.x) * blockDim.x;

    // Unrolling reduction within a block, with boundary check
    if (idx + 3 * blockDim.x < n) {
        g_idata[idx] += g_idata[idx + blockDim.x];
        g_idata[idx] += g_idata[idx + 2 * blockDim.x];
        g_idata[idx] += g_idata[idx + 3 * blockDim.x];
    }

    __syncthreads();

    // Complete reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}