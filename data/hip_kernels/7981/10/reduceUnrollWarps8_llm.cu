#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrollWarps8(int *g_idata, int *g_odata, unsigned int n){
    // Calculate thread index and corresponding data index
    unsigned int tid = threadIdx.x;
    unsigned int idx = (8 * blockIdx.x) * blockDim.x + threadIdx.x;
    int *idata = g_idata + (8 * blockIdx.x) * blockDim.x;

    // Unroll the loop to handle 8 contiguous elements
    if (idx + 7 * blockDim.x < n) {
        int sum = g_idata[idx];
        sum += g_idata[idx + blockDim.x];
        sum += g_idata[idx + 2 * blockDim.x];
        sum += g_idata[idx + 3 * blockDim.x];
        sum += g_idata[idx + 4 * blockDim.x];
        sum += g_idata[idx + 5 * blockDim.x];
        sum += g_idata[idx + 6 * blockDim.x];
        sum += g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = sum;
    }
    __syncthreads();

    // Reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // Last warp unrolled manually
    if (tid < 32) {
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}