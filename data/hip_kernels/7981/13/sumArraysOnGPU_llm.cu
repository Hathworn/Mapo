#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N){
    // Use shared memory to reduce global memory accesses
    extern __shared__ float sharedMem[];
    float *sharedA = sharedMem;
    float *sharedB = sharedMem + blockDim.x;

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N) {
        // Load data from global memory to shared memory
        sharedA[threadIdx.x] = A[i];
        sharedB[threadIdx.x] = B[i];
        __syncthreads();

        // Perform the computation
        C[i] = sharedA[threadIdx.x] + sharedB[threadIdx.x];
    }
}