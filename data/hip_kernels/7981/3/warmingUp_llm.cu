#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmingUp(int *g_idata, int *g_odata, unsigned int n) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (idx >= n) return;

    // Shared memory for better data access
    extern __shared__ int smem[];
    smem[tid] = g_idata[idx];
    __syncthreads();

    // Use shared memory to perform reduction
    for (int stride = 1; stride < blockDim.x; stride <<= 1) {
        int index = 2 * stride * tid;
        if (index < blockDim.x) {
            smem[index] += smem[index + stride];
        }
        __syncthreads();
    }

    // Write result to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = smem[0];
    }
}