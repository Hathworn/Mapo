#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling2(int *g_idata, int *g_odata, unsigned int n) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = (2 * blockIdx.x) * blockDim.x + threadIdx.x;

    int *idata = g_idata + (2 * blockIdx.x) * blockDim.x;

    // Use shared memory to reduce global memory accesses
    extern __shared__ int sdata[];
    sdata[tid] = (idx < n) ? g_idata[idx] : 0;
    if (idx + blockDim.x < n) sdata[tid] += g_idata[idx + blockDim.x];

    __syncthreads();

    // Perform the reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}