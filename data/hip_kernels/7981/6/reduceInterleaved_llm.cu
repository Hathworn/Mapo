#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceInterleaved(int *g_idata, int *g_odata, unsigned int n) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int *idata = g_idata + blockIdx.x * blockDim.x;

    if (idx >= n) return;

    // Use shared memory for faster access within a block
    extern __shared__ int sdata[];
    sdata[tid] = idata[tid];
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            // Perform warp shuffle reduction
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}