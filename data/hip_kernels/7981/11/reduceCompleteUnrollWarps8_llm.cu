#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceCompleteUnrollWarps8(int *g_idata, int *g_odata, unsigned int n) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = (8 * blockIdx.x) * blockDim.x + threadIdx.x;
    int *idata = g_idata + (8 * blockIdx.x) * blockDim.x;

    // Unroll the reduction step
    if (idx + 7 * blockDim.x < n) {
        int sum = g_idata[idx];
        sum += g_idata[idx + blockDim.x];
        sum += g_idata[idx + 2 * blockDim.x];
        sum += g_idata[idx + 3 * blockDim.x];
        sum += g_idata[idx + 4 * blockDim.x];
        sum += g_idata[idx + 5 * blockDim.x];
        sum += g_idata[idx + 6 * blockDim.x];
        sum += g_idata[idx + 7 * blockDim.x];
        idata[tid] = sum;
    }
    __syncthreads();

    // Perform intra-block reduction using unrolling
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();
    
    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();
    
    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();
    
    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();

    if (tid < 32) {
        volatile int *vmem = idata; // Use volatile to prevent optimization issues
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}