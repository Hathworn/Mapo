#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling8(int *g_idata, int *g_odata, unsigned int n) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = (8 * blockIdx.x) * blockDim.x + threadIdx.x;

    int *idata = g_idata + (8 * blockIdx.x) * blockDim.x;

    // Load data and perform unrolling
    int sum = 0;
    if (idx + 7 * blockDim.x < n) {
        sum = g_idata[idx];
        sum += g_idata[idx + blockDim.x];
        sum += g_idata[idx + 2 * blockDim.x];
        sum += g_idata[idx + 3 * blockDim.x];
        sum += g_idata[idx + 4 * blockDim.x];
        sum += g_idata[idx + 5 * blockDim.x];
        sum += g_idata[idx + 6 * blockDim.x];
        sum += g_idata[idx + 7 * blockDim.x];
    }
    idata[tid] = sum;
    __syncthreads();

    // Reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}