#include "hip/hip_runtime.h"
```c++
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighboredLess(int *g_idata, int *g_odata, unsigned int n) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ int sdata[];  // Declare shared memory
    if (idx < n) {
        sdata[tid] = g_idata[idx];  // Load input into shared memory
    } else {
        sdata[tid] = 0;  // Deal with out of bound threads
    }
    __syncthreads();  // Ensure all data is loaded

    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];  // Perform reduction in shared memory
        }
        __syncthreads();  // Ensure all adds at one stage are done before the next
    }

    if (tid == 0) g_odata[blockIdx.x] = sdata[0];  // Write result for this block to global memory
}