#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, const int nx, const int ny)
{
    // Use shared memory to reduce global memory accesses
    __shared__ float sA[32][32];
    __shared__ float sB[32][32];

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < nx && iy < ny) 
    {
        int idx = iy * nx + ix;
        sA[threadIdx.y][threadIdx.x] = A[idx];
        sB[threadIdx.y][threadIdx.x] = B[idx];
        
        __syncthreads(); // Ensure all data is loaded into shared memory
        
        // Perform addition in shared memory
        C[idx] = sA[threadIdx.y][threadIdx.x] + sB[threadIdx.y][threadIdx.x];
    }
}