#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighbored(int *g_idata, int *g_odata, unsigned int n) {
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return;

    extern __shared__ int sdata[];
    sdata[tid] = g_idata[idx];
    __syncthreads();

    // Perform reduction using shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}