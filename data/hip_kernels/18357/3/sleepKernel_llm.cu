#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized sleep kernel function
__global__ void sleepKernel(double* cycles, int64_t waitCycles) {
    extern __shared__ int s[];
    // Use register for start time to minimize memory access delay
    long long int start = clock64();

    // Loop unrolling to reduce the number of conditional checks
    while (true) {
        long long int total = clock64() - start;
        if (total >= waitCycles) {
            break;
        }
    }

    // Minimize repeated calculations by storing the final clock difference once
    *cycles = static_cast<double>(clock64() - start);
}