#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void signedGPU(int numTests, int* ns, int* ds, int* qs, int* rs) {
    // Get global thread index for optimized access
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check index to prevent out-of-bounds access
    if (i < numTests) {
        int n = ns[i];
        int d = ds[i];

        // Perform division and modulus operations
        qs[i] = n / d;
        rs[i] = n % d;
    }
}
```
