#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unsignedGPU(int numTests, unsigned* ns, unsigned* ds, unsigned* qs, unsigned* rs) {
    // Use thread index to allow parallel execution
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numTests) { // Ensure valid index range
        unsigned n = ns[i];
        unsigned d = ds[i];

        qs[i] = n / d;
        rs[i] = n % d;
    }
}