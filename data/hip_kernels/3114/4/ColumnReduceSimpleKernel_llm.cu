#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ColumnReduceSimpleKernel(const float* __restrict__ in, float* __restrict__ out, int num_planes, int num_rows, int num_cols) {
    // Use shared memory for reduced memory access latency
    extern __shared__ float shared_mem[];

    const int gid = threadIdx.x + blockIdx.x * blockDim.x;
    const int elems_per_plane = num_rows * num_cols;

    const int plane = gid / num_cols;
    const int col = gid % num_cols;

    if (plane >= num_planes)
        return;

    // Initialize sum in shared memory
    float* sum = shared_mem + threadIdx.x;
    *sum = 0.0f;

    // Unroll the loop for the first two rows
    *sum += in[plane * elems_per_plane + col] + in[plane * elems_per_plane + num_cols + col];

    // Perform reduction using shared memory
    for (int row = 2; row < num_rows; ++row) {
        *sum += in[plane * elems_per_plane + row * num_cols + col];
    }

    // Write the result to global memory
    out[plane * num_cols + col] = *sum;
}