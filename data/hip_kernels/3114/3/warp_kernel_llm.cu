#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warp_kernel(float* out, const float* in, const int* index, const float* weights, const int npixels, const int nchannels) {
    // Calculate linear index for pixel and channel
    int pixel = blockIdx.x * blockDim.x + threadIdx.x;
    int channel = blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary check
    if (channel < nchannels && pixel < npixels) {
        // Optimize memory access and calculations
        int pixelOffset = 4 * pixel;
        int outOffset = nchannels * pixel + channel;
        float result = 0.0f;
        #pragma unroll 4 // Unroll loop for performance
        for (int i = 0; i < 4; ++i) {
            int inOffset = nchannels * index[pixelOffset + i] + channel;
            result += in[inOffset] * weights[pixelOffset + i];
        }
        out[outOffset] = result;
    }
}