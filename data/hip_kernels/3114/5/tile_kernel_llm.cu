#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tile_kernel(const float* in, float* out, int num_planes, int num_rows, int num_cols) {
    // Calculate global thread ID
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    const int elems_per_plane = num_rows * num_cols;
    
    // Calculate plane and row indices
    const int plane = gid / num_rows;
    const int row = gid % num_rows;
    
    // Early exit if plane index is out of bounds
    if (plane >= num_planes) return;
    
    // Copy the entire row in one go
    for (int col = 0; col < num_cols; ++col) {
        // Use of shared memory buffer or coalesced memory access could be optimized here
        out[plane * elems_per_plane + row * num_cols + col] = in[plane * num_cols + col];
    }
}