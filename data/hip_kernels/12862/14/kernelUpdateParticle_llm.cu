#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelUpdateParticle(double *positions, double *velocities, double *pBests, double *gBest, int particlesCount, int dimensionsCount, double r1, double r2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Reduce the number of checks by using a smaller index range
    if (i < particlesCount * dimensionsCount) {
        // Use temporary variables to minimize memory access
        double position = positions[i];
        double velocity = velocities[i];
        double pBest = pBests[i];
        double gBestValue = gBest[i % dimensionsCount];

        // Compute new velocity based on update rule
        velocity = d_OMEGA * velocity + r1 * (pBest - position)
                                    + r2 * (gBestValue - position);

        // Update particle position
        positions[i] = position + velocity;
        velocities[i] = velocity;
    }
}