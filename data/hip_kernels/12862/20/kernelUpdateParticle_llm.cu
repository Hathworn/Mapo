#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelUpdateParticle(double *positions, double *velocities, double *pBests, double *gBest, int particlesCount, int dimensionsCount, double r1, double r2)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < particlesCount * dimensionsCount) {
        // Calculate linear index
        int particle = idx / dimensionsCount;
        int dimension = idx % dimensionsCount;

        // Compute velocity for current dimension
        double currentPos = positions[idx];
        double personalBest = pBests[idx];
        double globalBest = gBest[dimension];

        velocities[idx] = d_OMEGA * velocities[idx] + r1 * (personalBest - currentPos)
                        + r2 * (globalBest - currentPos);

        // Update particle position
        positions[idx] += velocities[idx];
    }
}