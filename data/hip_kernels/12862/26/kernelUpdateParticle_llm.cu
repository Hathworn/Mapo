#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelUpdateParticle(double *positions, double *velocities, double *pBests, double *gBest, int particlesCount, int dimensionsCount, double r1, double r2)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i >= particlesCount * dimensionsCount) // Check bounds to avoid unnecessary operations
        return;
    
    // Cache data in local variables for reduced global memory access
    double pos = positions[i];
    double vel = velocities[i];
    double pBest = pBests[i];
    double gBestVal = gBest[i % dimensionsCount];
    
    // Update velocities
    vel = d_OMEGA * vel + r1 * (pBest - pos) + r2 * (gBestVal - pos);
    
    // Update position
    pos += vel;
    
    // Store results back to global memory
    velocities[i] = vel;
    positions[i] = pos;
}