#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;
    // Load h/32 into register to avoid repeated computation
    int hDiv32 = h / 32;
    for (int y = 0; y < hDiv32; y++) {
        for (int x = 0; x < w; x += 16) {
            // Precompute the base index to minimize computation inside loops
            int baseIdx = y * w + x;
            unsigned int val = minmax[baseIdx + tx];
            if (val) {
                for (int k = 0; k < 32 && numPts < maxPts; k++) {
                    if (val & 0x1) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}