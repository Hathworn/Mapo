#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Calculate global thread coordinates
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate the size of a 2D slice
    int sz = height * pitch;

    // Calculate the position index for the current thread in the 3D array
    int p = blockIdx.z * sz + y * pitch + x;

    // Ensure thread is within bounds before performing operations
    if (x < width && y < height) {
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
}