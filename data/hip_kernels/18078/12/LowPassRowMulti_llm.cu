#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LowPassRowMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    __shared__ float data[CONVROW_W + 2*RADIUS];
    const int tx = threadIdx.x;
    const int block = blockIdx.x / (NUM_SCALES + 3);
    const int scale = blockIdx.x - (NUM_SCALES + 3) * block;
    const int xout = block * CONVROW_W + tx;
    const int loadPos = xout - RADIUS;
    const int yptr = blockIdx.y * pitch;
    const int writePos = yptr + height * pitch * scale + xout;
    float *kernel = d_Kernel + scale * 16;

    // Load data into shared memory with boundary checks
    if (loadPos < 0)
        data[tx] = d_Data[yptr];
    else if (loadPos >= width)
        data[tx] = d_Data[yptr + width - 1];
    else
        data[tx] = d_Data[yptr + loadPos];
    __syncthreads();

    // Compute convolution, only if within bounds
    if (xout < width && tx < CONVROW_W) {
        float result = 0.0f;
        #pragma unroll 5 // Unroll loop for better performance
        for (int i = 0; i < 5; ++i) {
            result += (data[tx + i] + data[tx + (8 - i)]) * kernel[i];
        }
        d_Result[writePos] = result;
    }
    __syncthreads();
}