#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    // Define thread index once
    const int tx = threadIdx.x;

    // Use shared memory for numPts
    __shared__ int numPtsShared;
    numPtsShared = 0;

    // Convert loop limits to depend on blockIdx.y for parallel blocks
    for (int y = blockIdx.y; y < h / 32; y += gridDim.y) {
        for (int x = blockIdx.x * 16 + tx; x < w; x += blockDim.x * 16) {
            unsigned int val = minmax[y * w + x];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    // Use atomic operation to safely update numPtsShared
                    if (val & 0x1 && numPtsShared < maxPts) {
                        unsigned int index = atomicAdd(&numPtsShared, 1);
                        if (index < maxPts) {  // Ensure index is within range
                            ptrs[16 * index + tx] = (y * 32 + k) * w + x;
                        }
                    }
                    val >>= 1;
                }
            }
        }
    }
}