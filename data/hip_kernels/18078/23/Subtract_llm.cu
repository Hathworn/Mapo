#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Calculate coordinates using block size and thread index
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    // Calculate the linear index
    int p = y * width + x;
    
    // Only subtract if within bounds
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    
    // Synchronization not needed here, as there's no shared memory use
}