#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int pitch, int height)
{
    // Optimizing shared memory usage
    __shared__ float data1[3*(MINMAX_W + 2)];
    __shared__ float data2[3*(MINMAX_W + 2)];
    __shared__ float data3[3*(MINMAX_W + 2)];
    __shared__ float ymin[(MINMAX_W + 2)];
    __shared__ float ymax[(MINMAX_W + 2)];

    // Current tile and apron limits, relative to row start
    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_W;
    const int xEndClamped = min(xStart + MINMAX_W - 1, width - 1);
    const int memWid = MINMAX_W + 2;
    
    int xReadPos = xStart + tx - WARP_SIZE;
    int xWritePos = xStart + tx;
    int memPos0 = tx - WARP_SIZE + 1;
    int memPos1 = memPos0;
    int yq = 0;
    unsigned int output = 0;

    for (int y = 0; y < 32 + 2; y++) {
        output >>= 1;
        int memPos = yq * memWid + memPos0;
        int yp = max(32 * blockIdx.y + y - 1, 0);
        yp = min(yp, height - 1);
        int readStart = yp * pitch;

        // Efficient data loading
        if (tx >= (WARP_SIZE - 1)) {
            float val1 = (xReadPos < 0 || xReadPos >= width) ? 0 : d_Data1[readStart + xReadPos];
            float val2 = (xReadPos < 0 || xReadPos >= width) ? 0 : d_Data2[readStart + xReadPos];
            float val3 = (xReadPos < 0 || xReadPos >= width) ? 0 : d_Data3[readStart + xReadPos];
            data1[memPos] = val1;
            data2[memPos] = val2;
            data3[memPos] = val3;
        }
        __syncthreads();

        int memPos2 = yq * memWid + tx;
        if (y > 1 && tx < memWid) {
            // Combine computations to minimize shared memory accesses
            float min1 = fminf(fminf(data1[memPos0], data1[memPos1]), data1[memPos2]);
            float min2 = fminf(fminf(data2[memPos0], data2[memPos1]), data2[memPos2]);
            float min3 = fminf(fminf(data3[memPos0], data3[memPos1]), data3[memPos2]);
            float max1 = fmaxf(fmaxf(data1[memPos0], data1[memPos1]), data1[memPos2]);
            float max2 = fmaxf(fmaxf(data2[memPos0], data2[memPos1]), data2[memPos2]);
            float max3 = fmaxf(fmaxf(data3[memPos0], data3[memPos1]), data3[memPos2]);
            ymin[tx] = fminf(fminf(min1, min2), min3);
            ymax[tx] = fmaxf(fmaxf(max1, max2), max3);
        }
        __syncthreads();

        if (y > 1 && tx < MINMAX_W && xWritePos <= xEndClamped) {
            // Simplify min and max calculations
            float minv = fminf(fminf(fminf(ymin[tx], ymin[tx + 2]), data2[memPos0 + 1]), data2[memPos2 + 1]);
            minv = fminf(minv, d_Threshold[1]);
            float maxv = fmaxf(fmaxf(fmaxf(ymax[tx], ymax[tx + 2]), data2[memPos0 + 1]), data2[memPos2 + 1]);
            maxv = fmaxf(maxv, d_Threshold[0]);
            if (data2[memPos1 + 1] < minv || data2[memPos1 + 1] > maxv) {
                output |= 0x80000000;
            }
        }
        __syncthreads();

        memPos0 = memPos1;
        memPos1 = memPos2;
        yq = (yq < 2 ? yq + 1 : 0);
    }

    if (tx < MINMAX_W && xWritePos < width) {
        int writeStart = blockIdx.y * pitch + xWritePos;
        d_Result[writeStart] = output;
    }
}