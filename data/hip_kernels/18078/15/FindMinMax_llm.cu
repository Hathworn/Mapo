#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * blockDim.x + tx;
    const int y = blockIdx.y * 16;
    int p = y * pitch + x;

    float local_min = FLT_MAX;
    float local_max = -FLT_MAX;

    // Ensure threads within bounds to avoid reading invalid memory
    for (int ty = 0; ty < 16 && y + ty < height; ty++) {
        float val = (x < width) ? d_Data[p] : d_Data[p - x];
        local_min = fminf(local_min, val);
        local_max = fmaxf(local_max, val);
        p += pitch;
    }

    minvals[tx] = local_min;
    maxvals[tx] = local_max;
    __syncthreads();

    // Reduction to find min/max using binary tree approach
    for (int d = blockDim.x / 2; d > 0; d >>= 1) {
        if (tx < d) {
            minvals[tx] = fminf(minvals[tx], minvals[tx + d]);
            maxvals[tx] = fmaxf(maxvals[tx], maxvals[tx + d]);
        }
        __syncthreads();
    }

    // Store result in global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}