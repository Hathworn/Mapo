#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    // Calculate global thread index
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * pitch + x;

    // Ensure within bounds before computation
    if (x < width && y < height) {
        // Perform subtraction without __syncthreads (not needed here)
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}