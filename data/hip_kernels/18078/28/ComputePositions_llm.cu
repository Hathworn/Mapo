#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimized for better performance and readability
__global__ void ComputePositions(float *g_Data1, float *g_Data2, float *g_Data3, int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
    // Compute the global index for the current thread
    int i = blockIdx.x * POSBLK_SIZE + threadIdx.x;

    // Return if index exceeds number of points
    if (i >= numPts) return;

    // Get position from d_Ptrs
    int p = d_Ptrs[i];

    // Pre-calculate indices for efficient memory access
    int pMinus1 = p - 1;
    int pPlus1 = p + 1;
    int pMinusW = p - w;
    int pPlusW = p + w;

    // Load necessary data into registers
    float val[7];
    val[0] = g_Data2[p];
    val[1] = g_Data2[pMinus1];
    val[2] = g_Data2[pPlus1];
    val[3] = g_Data2[pMinusW];
    val[4] = g_Data2[pPlusW];
    val[5] = g_Data3[p];
    val[6] = g_Data1[p];

    // Calculate derivatives
    float dx = 0.5f * (val[2] - val[1]);
    float dxx = 2.0f * val[0] - val[1] - val[2];
    float dy = 0.5f * (val[4] - val[3]);
    float dyy = 2.0f * val[0] - val[3] - val[4];
    float ds = 0.5f * (val[6] - val[5]);
    float dss = 2.0f * val[0] - val[5] - val[6];

    // Calculate mixed derivatives
    float dxy = 0.25f * (g_Data2[pPlusW+1] + g_Data2[pMinusW-1] - g_Data2[pMinusW+1] - g_Data2[pPlusW-1]);
    float dxs = 0.25f * (g_Data3[pPlus1] + g_Data1[pMinus1] - g_Data1[pPlus1] - g_Data3[pMinus1]);
    float dys = 0.25f * (g_Data3[pPlusW] + g_Data1[pMinusW] - g_Data3[pMinusW] - g_Data1[pPlusW]);

    // Calculate determinant and inverse determinant
    float idxx = dyy * dss - dys * dys;
    float idxy = dys * dxs - dxy * dss;
    float idxs = dxy * dys - dyy * dxs;
    float idyy = dxx * dss - dxs * dxs;
    float idys = dxy * dxs - dxx * dys;
    float idss = dxx * dyy - dxy * dxy;
    float det = idxx * dxx + idxy * dxy + idxs * dxs;
    float idet = 1.0f / det;

    // Compute position adjustments
    float pdx = idet * (idxx * dx + idxy * dy + idxs * ds);
    float pdy = idet * (idxy * dx + idyy * dy + idys * ds);
    float pds = idet * (idxs * dx + idys * dy + idss * ds);

    // Check bounds for position adjustments
    if (pdx < -0.5f || pdx > 0.5f || pdy < -0.5f || pdy > 0.5f || pds < -0.5f || pds > 0.5f)
    {
        pdx = __fdividef(dx, dxx);
        pdy = __fdividef(dy, dyy);
        pds = __fdividef(ds, dss);
    }

    // Calculate sift details
    float dval = 0.5f * (dx * pdx + dy * pdy + ds * pds);
    d_Sift[i + 0 * maxPts] = (p % w) + pdx;
    d_Sift[i + 1 * maxPts] = (p / w) + pdy;
    d_Sift[i + 2 * maxPts] = d_ConstantA[0] * exp2f(pds * d_ConstantB[0]);
    d_Sift[i + 3 * maxPts] = val[0] + dval;

    // Compute trace and determinant for feature
    float tra = dxx + dyy;
    det = dxx * dyy - dxy * dxy;
    d_Sift[i + 4 * maxPts] = __fdividef(tra * tra, det);
}