#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define WARP_SIZE 32 // # of threads that are executed together (constant valid on most hardware)

// Optimize kernel to use shared memory and efficient indexing
__global__ void _cuda_add_scalar(int *in, int scalar, int n)
{
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = globalIdx; i < n; i += stride) {
        // Adding scalar value to each element of the input array 
        in[i] += scalar;
    }
}