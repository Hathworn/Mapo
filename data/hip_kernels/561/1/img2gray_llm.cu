#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput) {
    // Calculate row and col using thread and block indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds
    if (row < height && col < width) {
        // Calculate pixel position
        int pos = (row * width + col) * Channels;
        
        // Convert pixel to grayscale using fixed point arithmetic
        imageOutput[row * width + col] = 
            static_cast<unsigned char>(
                (__float2uint_rz(imageInput[pos + RED] * 77) +  // 0.299 * 256 = 76.544 (approx 77)
                 __float2uint_rz(imageInput[pos + GREEN] * 150) +  // 0.587 * 256 = 150.272 (approx 150)
                 __float2uint_rz(imageInput[pos + BLUE] * 29)) >> 8); // 0.114 * 256 = 29.184 (approx 29)
    }
}