#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t

__global__ void CorrelateData_1d(const int nthreads, int num, int topwidth, int topheight, int topchannels, int topcount, int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int kernel_size, int stride1, int stride2, int bottomwidth, int bottomheight, int bottomchannels, const float *bottom0, const float *bottom1, float *top)
{
    extern __shared__ float patch_data[];

    // Compute position of kernel upper-left corner in image 1
    int x1 = blockIdx.x * stride1 + max_displacement;
    int y1 = blockIdx.y * stride1;
    int item = blockIdx.z;
    int ch_off = threadIdx.x;

    // Load 3D patch into shared memory - optimized loop order to improve memory access patterns
    for (int ch = ch_off; ch < bottomchannels; ch += THREADS_PER_WARP) { // CHANNELS
        for (int j = 0; j < kernel_size; j++) { // HEIGHT
            for (int i = 0; i < kernel_size; i++) { // WIDTH
                int idx1 = ((item * bottomheight + y1 + j) * bottomwidth + x1 + i) * bottomchannels + ch;
                int idxPatchData = ((j * kernel_size + i) * bottomchannels) + ch;
                patch_data[idxPatchData] = bottom0[idx1];
            }
        }
    }

    __syncthreads();

    float sum = 0;

    // Compute correlation - optimized to use local memory for sum
    for (int top_channel = 0; top_channel < topchannels; top_channel++) {
        int s2o = (top_channel % neighborhood_grid_width + x_shift) * stride2;
        int x2 = x1 + s2o;

        for (int ch = ch_off; ch < bottomchannels; ch += THREADS_PER_WARP) { // CHANNELS
            for (int j = 0; j < kernel_size; j++) { // HEIGHT
                for (int i = 0; i < kernel_size; i++) { // WIDTH
                    int idxPatchData = ((j * kernel_size + i) * bottomchannels) + ch;
                    int idx2 = ((item * bottomheight + y1 + j) * bottomwidth + x2 + i) * bottomchannels + ch;
                    sum += patch_data[idxPatchData] * bottom1[idx2];
                }
            }
        }

        __syncthreads();

        if (ch_off == 0) {
            float total_sum = 0;
            total_sum += sum;
            const int sumelems = kernel_size * kernel_size * bottomchannels;
            const int index = ((top_channel * topheight + blockIdx.y) * topwidth) + blockIdx.x;
            top[index + item * topcount] = total_sum / (float)sumelems;
        }
    }
}