#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blob_rearrange_kernel2(const float *in, float *out, int num, int channels, int width, int height, int widthheight, int padding, int pwidthheight)
{
    // Calculate global index
    int xy = blockIdx.x * blockDim.x + threadIdx.x;
    if (xy >= widthheight) return;

    int ch = blockIdx.y;
    int n  = blockIdx.z;

    // Directly assign the value without additional __syncthreads() as it's not needed here
    float value = in[(n * channels + ch) * widthheight + xy];
    
    // Precompute index for output to optimize memory access
    int xpad = xy % width + padding;
    int ypad = xy / width + padding;
    int xypad = ypad * (width + 2 * padding) + xpad;

    out[(n * pwidthheight + xypad) * channels + ch] = value;
}