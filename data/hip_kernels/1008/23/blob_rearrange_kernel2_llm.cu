#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blob_rearrange_kernel2(const float *in, float *out, int num, int channels, int width, int height, int widthheight, int padding, int pwidthheight)
{
    int xy = blockIdx.x * blockDim.x + threadIdx.x;
    if (xy >= widthheight)
        return;

    int ch = blockIdx.y;
    int n = blockIdx.z;
    
    // Removed synchronization as it is not needed here
    float value = in[(n * channels + ch) * widthheight + xy];

    int xpad = (xy % width + padding);
    int ypad = (xy / width + padding);
    int xypad = ypad * (width + 2 * padding) + xpad;

    out[(n * pwidthheight + xypad) * channels + ch] = value;
}