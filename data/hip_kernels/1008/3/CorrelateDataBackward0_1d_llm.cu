#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t

// == Dimension rearrangement Kernel

__global__ void CorrelateDataBackward0_1d(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels, int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2, int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size, float *bottom0diff, const float *bottom1, const float *topdiff)
{
    CUDA_KERNEL_LOOP(index, nthreads) {
        int n = index % bottomchannels; //channels
        int l = (index / bottomchannels) % bottomwidth + pad_size; //w-pos
        int m = (index / bottomchannels / bottomwidth) % bottomheight; //h-pos

        const int round_off = ROUND_OFF;
        const int round_off_s1 = stride1 * round_off;

        // Calculations modified to use integer arithmetic and prevent redundant computations
        int xmin = max(0, (l - 2*kernel_radius - max_displacement + round_off_s1 - 1) / stride1 + 1 - round_off);
        int ymin = max(0, (m - 2*kernel_radius + round_off_s1 - 1) / stride1 + 1 - round_off);
        int xmax = min(topwidth - 1, (l - max_displacement + round_off_s1) / stride1 - round_off);
        int ymax = min(topheight - 1, (m + round_off_s1) / stride1 - round_off);

        float sum = 0;
        if (xmax >= 0 && ymax >= 0 && xmin <= topwidth - 1 && ymin <= topheight - 1) {
            for (int o = x_shift; o < x_shift + neighborhood_grid_width; o++) {
                int s2o = stride2 * o;
                int idxbot1 = ((item * pbottomheight + m) * pbottomwidth + (l+s2o)) * bottomchannels + n;
                float bot1tmp = bottom1[idxbot1];

                int op = (o-x_shift);
                int idxopoffset = (item * topchannels + op);

                for (int y = ymin; y <= ymax; y++) {
                    for (int x = xmin; x <= xmax; x++) {
                        int idxtopdiff = (idxopoffset * topheight + y) * topwidth + x;
                        sum += topdiff[idxtopdiff] * bot1tmp;
                    }
                }
            }
        }
        const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
        const int bot0index = ((n * bottomheight) + m) * bottomwidth + (l-pad_size);
        bottom0diff[bot0index + item*bottomcount] = sum / (float)sumelems;
    }
}