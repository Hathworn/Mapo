#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t

// == Dimension rearrangement Kernel

__global__ void CorrelateDataSubtract_1d(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels, int topcount, int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2, int bottomwidth, int bottomheight, int bottomchannels, const float *bottom0, const float *bottom1, float *top)
{
    CUDA_KERNEL_LOOP(index, nthreads) {
        int x = index % topwidth; // w-pos
        int y = (index / topwidth) % topheight; // h-pos
        int c = (index / topwidth / topheight) % topchannels; // channels
        
        // Pre-calculate shared indices 
        int x1 = x * stride1 + kernel_radius + max_displacement;
        int y1 = y * stride1 + kernel_radius;
        int s2o = (c % neighborhood_grid_width + x_shift) * stride2;
        int x2 = x1 + s2o;

        float sum = 0.0f; // Use float precision directly
        const int sumelems = (kernel_radius * 2 + 1) * (kernel_radius * 2 + 1) * bottomchannels;
        
        // Iterate efficiently over 3D patch
        #pragma unroll // Unroll the innermost loop for better performance
        for(int l = 0; l < bottomchannels; l++) { // CHANNELS
            for(int j = -kernel_radius; j <= kernel_radius; j++) { // HEIGHT
                for(int i = -kernel_radius; i <= kernel_radius; i++) { // WIDTH
                    // Calculate indices for bottom data access
                    int idx1 = ((item * bottomheight + y1 + j) * bottomwidth + x1 + i) * bottomchannels + l;
                    int idx2 = ((item * bottomheight + y1 + j) * bottomwidth + x2 + i) * bottomchannels + l;
                    
                    // Perform correlation and accumulate result
                    sum += fabsf(bottom0[idx1] - bottom1[idx2]);
                }
            }
        }
        // Store result in the output
        top[index + item * topcount] = sum / (float)sumelems;
    }
}