#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) ((n) + (t) - 1) / (t)

// == Dimension rearrangement Kernel

__global__ void CorrelateDataBackward1Subtract_1d(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels, 
                                                   int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, 
                                                   int stride1, int stride2, int bottomwidth, int bottomheight, int pbottomwidth, 
                                                   int pbottomheight, int bottomchannels, int bottomcount, int pad_size, 
                                                   const float *bottom0, const float *bottom1, float *bottom1diff, const float *topdiff)
{
    CUDA_KERNEL_LOOP(index, nthreads) {
        
        int l = index % bottomwidth + pad_size; // w-pos
        int m = (index / bottomwidth) % bottomheight; // h-pos
        int n = (index / bottomwidth / bottomheight) % bottomchannels; // channels

        // Precompute some values to reduce repeated calculations
        const int round_off = ROUND_OFF;
        const int round_off_s1 = stride1 * round_off;
        const int m_offset = m * pbottomwidth;
        const int idxBase = ((item * pbottomheight + m) * pbottomwidth + l);

        float sum = 0;
        for(int o = x_shift; o < x_shift + neighborhood_grid_width; o++) {

            int s2o = stride2 * o;
            int idxBotO = idxBase - s2o;

            // Calculate ranges and clamp them
            int xmin = max(0, (l - 2 * kernel_radius - max_displacement - s2o + round_off_s1 - 1) / stride1 + 1 - round_off);
            int ymin = max(0, (m - 2 * kernel_radius + round_off_s1 - 1) / stride1 + 1 - round_off);

            int xmax = min(topwidth - 1, (l - max_displacement - s2o + round_off_s1) / stride1 - round_off);
            int ymax = min(topheight - 1, (m + round_off_s1) / stride1 - round_off);

            if (xmax >= xmin && ymax >= ymin) {

                float bot0tmp = bottom0[idxBotO * bottomchannels + n];
                float bot1tmp = bottom1[idxBotO * bottomchannels + n];
                float sign = (bot0tmp >= bot1tmp) ? float(-1.0) : float(1.0);
                
                int op = (o - x_shift); // index [o,p]
                int idxOpOffset = (item * topchannels + op);

                for (int y = ymin; y <= ymax; y++) {
                    int idxtopdiffBase = (idxOpOffset * topheight + y) * topwidth;
                    for (int x = xmin; x <= xmax; x++) {
                        int idxtopdiff = idxtopdiffBase + x; // topdiff[x,y,o,p]
                        sum += topdiff[idxtopdiff] * sign;
                    }
                }
            }
        }
        
        const int sumelems = (kernel_radius * 2 + 1) * (kernel_radius * 2 + 1) * bottomchannels;
        bottom1diff[index + item * bottomcount] = sum / (float)sumelems;
    }
}