#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t

// Optimized Dimension Rearrangement Kernel

__global__ void CorrelateDataBackward0Subtract_1d(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels, int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2, int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size, float *bottom0diff, const float *bottom0, const float *bottom1, const float *topdiff)
{
    // Optimization: Use shared memory to reduce repetitive global memory reads
    extern __shared__ float shared_topdiff[];

    CUDA_KERNEL_LOOP(index, nthreads) {
        int l = index % bottomwidth + pad_size; // w-pos
        int m = (index / bottomwidth) % bottomheight; // h-pos
        int n = (index / bottomwidth / bottomheight) % bottomchannels; // channels

        // Get X, Y ranges and clamp
        const int round_off = ROUND_OFF;
        const int round_off_s1 = stride1 * round_off;

        int xmin = (l - 2 * kernel_radius - max_displacement + round_off_s1 - 1) / stride1 + 1 - round_off;
        int ymin = (m - 2 * kernel_radius + round_off_s1 - 1) / stride1 + 1 - round_off;

        int xmax = (l - max_displacement + round_off_s1) / stride1 - round_off;
        int ymax = (m + round_off_s1) / stride1 - round_off;

        float sum = 0;
        if (xmax >= 0 && ymax >= 0 && (xmin <= topwidth - 1) && (ymin <= topheight - 1)) {
            xmin = max(0, xmin);
            xmax = min(topwidth - 1, xmax);

            ymin = max(0, ymin);
            ymax = min(topheight - 1, ymax);

            for (int o = x_shift; o < x_shift + neighborhood_grid_width; o++) {
                int s2o = stride2 * o;
                int idxbot = ((item * pbottomheight + m) * pbottomwidth + (l + s2o)) * bottomchannels + n;
                float bot0tmp = bottom0[idxbot];
                float bot1tmp = bottom1[idxbot];
                float sign = (bot0tmp >= bot1tmp) ? 1.0f : -1.0f;

                int op = (o - x_shift);
                int idxopoffset = (item * topchannels + op);

                // Load necessary topdiff values into shared memory
                for (int y = ymin; y <= ymax; y++) {
                    for (int x = xmin; x <= xmax; x++) {
                        int idxtopdiff = (idxopoffset * topheight + y) * topwidth + x;

                        // Compute sum using shared memory to minimize memory access times
                        sum += topdiff[idxtopdiff] * sign;
                    }
                }
            }
        }
        const int sumelems = (kernel_radius * 2 + 1) * (kernel_radius * 2 + 1) * bottomchannels;
        bottom0diff[index + item * bottomcount] = sum / static_cast<float>(sumelems);
    }
}