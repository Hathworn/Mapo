#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t

// == Dimension rearrangement Kernel

__global__ void CorrelateDataBackward1_1d(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels, int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2, int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size, const float *bottom0, float *bottom1diff, const float *topdiff)
{
    CUDA_KERNEL_LOOP(index, nthreads) {
        // Pre-calculate indices
        int n = index % bottomchannels; // channels
        int l = (index / bottomchannels) % bottomwidth + pad_size; // w-pos
        int m = (index / bottomchannels / bottomwidth) % bottomheight; // h-pos

        const int round_off = ROUND_OFF;
        const int round_off_s1 = stride1 * round_off;
        float sum = 0;

        for(int o = x_shift; o < x_shift + neighborhood_grid_width; o++) {
            int s2o = stride2 * o;

            // Calculate boundaries with clamping
            int xmin = max(0, (l - 2*kernel_radius - max_displacement - s2o + round_off_s1 - 1) / stride1 + 1 - round_off);
            int ymin = max(0, (m - 2*kernel_radius + round_off_s1 - 1) / stride1 + 1 - round_off);

            int xmax = min(topwidth-1, (l - max_displacement - s2o + round_off_s1) / stride1 - round_off);
            int ymax = min(topheight-1, (m + round_off_s1) / stride1 - round_off);

            if(xmax >= 0 && ymax >= 0 && xmin <= topwidth-1 && ymin <= topheight-1) {
                // Read bottom0 data
                int idxbot0 = ((item * pbottomheight + m) * pbottomwidth + (l - s2o)) * bottomchannels + n;
                float bot0tmp = bottom0[idxbot0];

                // Pre-calculate index offset for topdiff
                int op = (o - x_shift);
                int idxOpOffset = (item * topchannels + op);

                for(int y = ymin; y <= ymax; y++) {
                    for(int x = xmin; x <= xmax; x++) {
                        int idxtopdiff = (idxOpOffset * topheight + y) * topwidth + x;
                        sum += topdiff[idxtopdiff] * bot0tmp;
                    }
                }
            }
        }

        const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
        const int bot1index = ((n * bottomheight) + m) * bottomwidth + (l-pad_size);
        bottom1diff[bot1index + item*bottomcount] = sum / (float)sumelems;
    }
}