#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CorrelateData_1d(const int nthreads, int num, int topwidth, int topheight, int topchannels, 
                                 int topcount, int max_displacement, int x_shift, int neighborhood_grid_width, 
                                 int kernel_radius, int kernel_size, int stride1, int stride2, int bottomwidth, 
                                 int bottomheight, int bottomchannels, const float *bottom0, const float *bottom1, 
                                 float *top) {
    extern __shared__ char patch_data_char[];

    float *patch_data = (float *)patch_data_char;

    int x1 = blockIdx.x * stride1 + max_displacement;
    int y1 = blockIdx.y * stride1;
    int item = blockIdx.z;
    int ch_off = threadIdx.x;

    // Load 3D patch into shared memory
    for (int j = 0; j < kernel_size; j++) {
        for (int i = 0; i < kernel_size; i++) {
            int ji_off = ((j * kernel_size) + i) * bottomchannels;
            for (int ch = ch_off; ch < bottomchannels; ch += (WARPS_PER_BLOCK * THREADS_PER_WARP)) {
                int idx1 = ((item * bottomheight + y1 + j) * bottomwidth + x1 + i) * bottomchannels + ch;
                int idxPatchData = ji_off + ch;
                patch_data[idxPatchData] = bottom0[idx1];
            }
        }
    }

    __syncthreads();

    __shared__ float sum[WARPS_PER_BLOCK * THREADS_PER_WARP];

    // Compute correlation
    for (int top_channel = 0; top_channel < topchannels; top_channel++) {
        sum[ch_off] = 0;
        int s2o = (top_channel % neighborhood_grid_width + x_shift) * stride2;
        // Use loop unrolling for kernel computation
        #pragma unroll
        for (int j = 0; j < kernel_size; j++) {
            #pragma unroll
            for (int i = 0; i < kernel_size; i++) {
                int ji_off = ((j * kernel_size) + i) * bottomchannels;
                for (int ch = ch_off; ch < bottomchannels; ch += (WARPS_PER_BLOCK * THREADS_PER_WARP)) {
                    int x2 = x1 + s2o;
                    int idxPatchData = ji_off + ch;
                    int idx2 = ((item * bottomheight + y1 + j) * bottomwidth + x2 + i) * bottomchannels + ch;
                    sum[ch_off] += patch_data[idxPatchData] * bottom1[idx2];
                }
            }
        }

        __syncthreads();

        if (ch_off == 0) {
            float total_sum = 0;
            // Use parallel reduction for summation
            for (int idx = 0; idx < WARPS_PER_BLOCK * THREADS_PER_WARP; idx++) {
                total_sum += sum[idx];
            }
            const int sumelems = kernel_size * kernel_size * bottomchannels;
            const int index = ((top_channel * topheight + blockIdx.y) * topwidth) + blockIdx.x;
            top[index + item * topcount] = total_sum / (float)sumelems;
        }
    }
}