#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blob_rearrange_kernel2_1d(const float *in, float *out, int num, int channels, int width, int height, int widthheight, int padding, int pwidthheight)
{
    int xy = blockIdx.x*blockDim.x + threadIdx.x;

    // Ensure threads within valid range to eliminate unnecessary computation
    if(xy < widthheight) {
        int ch = blockIdx.y;
        int n  = blockIdx.z;

        // Optimize memory access pattern by computing indices first
        int xpad  = (xy % width + padding);
        int ypad  = (xy / width);
        int xypad = ypad * (width + 2 * padding) + xpad;

        // Access and assign values without syncing threads as it's unnecessary
        out[(n * pwidthheight + xypad) * channels + ch] = in[(n * channels + ch) * widthheight + xy];
    }
}