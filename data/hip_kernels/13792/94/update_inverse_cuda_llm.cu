#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NMAX 1024

__global__ static void update_inverse_cuda(float *Ainv, float *u, int N, int rowstride, int k)
{
    __shared__ float A_k[NMAX], u_shared[NMAX], Ainv_u[NMAX], Ainv_shared[NMAX];

    // Load data into shared memory
    A_k[threadIdx.x] = Ainv[k * rowstride + threadIdx.x];
    u_shared[threadIdx.x] = u[threadIdx.x];

    // First, compute k'th element of Ainv_u using reduction
    Ainv_u[threadIdx.x] = u_shared[threadIdx.x] * A_k[threadIdx.x];
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (threadIdx.x < offset)
        {
            Ainv_u[threadIdx.x] += Ainv_u[threadIdx.x + offset];
        }
        __syncthreads();
    }

    float prefact = -1.0f / (1.0f + Ainv_u[0]);

    // Loop over rows to update Ainv
    for (int row = 0; row < N; row++)
    {
        // Load row into shared memory
        Ainv_shared[threadIdx.x] = Ainv[row * rowstride + threadIdx.x];
        __syncthreads();

        // Compute row'th element of Ainv_u using reduction
        Ainv_u[threadIdx.x] = u_shared[threadIdx.x] * Ainv_shared[threadIdx.x];
        __syncthreads();

        for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
        {
            if (threadIdx.x < offset)
            {
                Ainv_u[threadIdx.x] += Ainv_u[threadIdx.x + offset];
            }
            __syncthreads();
        }

        // Update Ainv with the computed value
        if (threadIdx.x < rowstride)
        {
            Ainv[row * rowstride + threadIdx.x] =
                Ainv_shared[threadIdx.x] + prefact * Ainv_u[0] * A_k[threadIdx.x];
        }
    }
}