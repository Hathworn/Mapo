#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *a, size_t N)
{
    int tid = threadIdx.x;
    __shared__ float s[BS];
    float sum = 0.0f;
    int blocks = (N + BS - 1) / BS;

    for (int ib = 0; ib < blocks; ib++)
    {
        int off = ib * BS + tid;
        s[tid] = (off < N) ? a[off] : 0.0f;  // Use conditional assignment to handle out-of-bounds access

        __syncthreads();  // Ensure shared memory load is complete

        for (int skip = BS / 2; skip > 0; skip >>= 1)
        {
            if (tid < skip)
                s[tid] += s[tid + skip];
            __syncthreads();  // Synchronize to ensure all additions are complete
        }

        if (tid == 0) 
            sum += s[0];  // Only accumulate sum in the first thread of the block
    }

    if (tid == 0) 
        a[0] = sum;  // Only the first thread writes the result
}