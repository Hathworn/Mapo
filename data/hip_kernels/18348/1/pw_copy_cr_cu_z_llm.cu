#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_copy_cr_cu_z(const double *zin, double *dout, const int n) {
    // Calculate global thread index
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Bound checking to avoid out-of-bounds access
    if (igpt < n) {
        dout[igpt] = zin[2 * igpt];
    }
}