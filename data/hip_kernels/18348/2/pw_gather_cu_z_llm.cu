#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_gather_cu_z(double *pwcc, const double *c, const double scale, const int ngpts, const int *ghatmap) {

    // Calculate global thread index for 1D grid
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x + gridDim.x * (blockIdx.y);

    // Check if the thread should perform the calculation
    if (igpt < ngpts) {
        // Perform scaled assignment
        int map_idx = ghatmap[igpt];
        pwcc[2 * igpt] = scale * c[2 * map_idx];
        pwcc[2 * igpt + 1] = scale * c[2 * map_idx + 1];
    }
}