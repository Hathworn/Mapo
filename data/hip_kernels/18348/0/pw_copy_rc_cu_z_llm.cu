#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_copy_rc_cu_z(const double *din, double *zout, const int n) {
    // Calculate global index
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop to process all elements
    for (int idx = igpt; idx < n; idx += gridDim.x * blockDim.x) {
        zout[2 * idx] = din[idx];
        zout[2 * idx + 1] = 0.0e0;
    }
}