#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void pw_scatter_cu_z(double *c, const double *pwcc, const double scale, const int ngpts, const int nmaps, const int *ghatmap) {

    // Calculate global thread index
    int igpt = blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y) + threadIdx.x;

    if (igpt < ngpts) {
        int index = 2 * igpt;
        int mapIdx = ghatmap[igpt];
        
        // Store scaled values
        double scaledReal = scale * pwcc[index];
        double scaledImag = scale * pwcc[index + 1];
        
        // Map and write the scaled values
        c[2 * mapIdx] = scaledReal;
        c[2 * mapIdx + 1] = scaledImag;

        // Handle nmaps == 2 case without recalculations
        if (nmaps == 2) {
            int mapIdxOffset = ghatmap[igpt + ngpts];
            c[2 * mapIdxOffset] = scaledReal;
            c[2 * mapIdxOffset + 1] = -scaledImag;
        }
    }
}