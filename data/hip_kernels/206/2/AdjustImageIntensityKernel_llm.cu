#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 512
#define BLOCK_SIZE_HOUGH 360
#define STEP_SIZE 5
#define NUMBER_OF_STEPS 360/STEP_SIZE

// Circ mask kernel storage
__constant__ int maskKernelX[NUMBER_OF_STEPS];
__constant__ int maskKernelY[NUMBER_OF_STEPS];

// Function to set precalculated relative coordinates for circle boundary coordinates
__global__ void AdjustImageIntensityKernel(float *imgOut, float *imgIn, int width, int height, float lowin, float lowout, float scale)
{
    __shared__ float bufData[BLOCK_SIZE];

    // Get the index of pixel
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check that it's not out of bounds before loading
    if (index < width * height) {
        // Load data to shared variable
        bufData[threadIdx.x] = imgIn[index];

        // Compute adjusted intensity level
        float tempLevel = (bufData[threadIdx.x] - lowin) * scale + lowout;

        // Use local variable for boundary conditions
        float result;
        if (tempLevel < 0) {
            result = 0;
        } else if (tempLevel > 1) {
            result = 1;
        } else {
            result = tempLevel;
        }

        // Write data back
        imgOut[index] = result;
    }
}