#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 512
#define BLOCK_SIZE_HOUGH 360
#define STEP_SIZE 5
#define NUMBER_OF_STEPS 360/STEP_SIZE

// Circ mask kernel storage
__constant__ int maskKernelX[NUMBER_OF_STEPS];
__constant__ int maskKernelY[NUMBER_OF_STEPS];

// Function to set precalculated relative coordinates for circle boundary coordinates
__global__ void ImageScalingKernel(float *imgOut, float *imgIn, int width, int height)
{
    __shared__ float inData[BLOCK_SIZE];
    // Get the index of pixel
    const int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    // Load data to shared variable only if within bounds
    if (index < width * height) {
        inData[threadIdx.x] = imgIn[index];
        // Scale the image data
        imgOut[index] = inData[threadIdx.x] / 255.0f;
    }

    __syncthreads();
}