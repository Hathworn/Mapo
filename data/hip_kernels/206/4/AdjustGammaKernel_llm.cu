#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 512
#define BLOCK_SIZE_HOUGH 360
#define STEP_SIZE 5
#define NUMBER_OF_STEPS 360/STEP_SIZE

// Circ mask kernel storage
__constant__ int maskKernelX[NUMBER_OF_STEPS];
__constant__ int maskKernelY[NUMBER_OF_STEPS];

// Function to set precalculated relative coordinates for circle boundary coordinates
__global__ void AdjustGammaKernel(float *imgOut, float *imgIn, int width, int height, float gamma, float minVal, float maxVal)
{
    __shared__ float bufData[BLOCK_SIZE];

    // Get the index of pixel
    const int index = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    
    // Check that it's not out of bounds
    if (index < (height * width)) {
        // Load data to shared variable
        bufData[threadIdx.x] = imgIn[index];

        // Find the according multiplier
        float tempLevel = (bufData[threadIdx.x] - minVal) / maxVal;

        // Apply gamma correction
        tempLevel = powf(tempLevel, 1.0f / gamma);

        // Check that it's within required range and clamp
        tempLevel = fminf(fmaxf(tempLevel, 0.0f), 1.0f);

        // Write data back
        imgOut[index] = tempLevel;
    }
}