#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure single thread performs the division
    if (idx == 0) {
        *accuracy /= N;
    }
}