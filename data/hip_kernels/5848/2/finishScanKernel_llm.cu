#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define dfloat double

#define p_eps 1e-6
#define p_Nsamples 1
#define p_primaryWeight 2.f
#define p_intersectDelta 0.1f
#define p_shadowDelta 0.15f
#define p_projectDelta 1e-2
#define p_maxLevel 5
#define p_maxNrays (2<<p_maxLevel)
#define p_apertureRadius 20.f
#define NRANDOM 10000

hipEvent_t startTimer, endTimer;

__global__ void finishScanKernel(const int N, int *scanv, int *starts) {
    // Calculate global thread index
    int n = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread operates within bounds
    if (n < N) {
        // Load start value once per block from shared memory or register
        int start = starts[blockIdx.x]; 
        
        // Perform scan increment
        scanv[n + 1] += start;
    }
}