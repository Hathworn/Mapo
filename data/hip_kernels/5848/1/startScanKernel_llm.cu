#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// risky
#define dfloat double

#define p_eps 1e-6

#define p_Nsamples 1

// ratio of importance in sampling primary ray versus random rays
#define p_primaryWeight 2.f

#define p_intersectDelta 0.1f

#define p_shadowDelta 0.15f
#define p_projectDelta 1e-2

#define p_maxLevel 5
#define p_maxNrays (2<<p_maxLevel)
#define p_apertureRadius 20.f
#define NRANDOM 10000

hipEvent_t startTimer, endTimer;

__global__ void startScanKernel(const int N, const int *v, int *scanv, int *starts){

  extern __shared__ int sharedData[];
  int *s_v0 = sharedData;
  int *s_v1 = sharedData + BLOCKSIZE;

  int j = threadIdx.x;
  int b = blockIdx.x;
  int n = j + b * BLOCKSIZE;

  // Use a single block-wise load operation instead of two separate
  s_v0[j] = (n < N) ? v[n] : 0;

  int offset = 1;
  
  // Optimize loop by combining reduction and propagation stages
  for (; offset < BLOCKSIZE; offset *= 2) {
    __syncthreads();
    int value = (j >= offset) ? s_v0[j - offset] : 0;
    s_v1[j] = s_v0[j] + value;
    
    __syncthreads();
    // Swap pointers to avoid extra copy
    int *temp = s_v0;
    s_v0 = s_v1;
    s_v1 = temp;
  }

  if (n < N) {
    scanv[n + 1] = s_v0[j];
  }

  if (j == (BLOCKSIZE - 1)) {
    starts[b + 1] = s_v0[j];
  }
}