#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel0(int n, float a, float *x, float *y) {
    // Using grid-stride loop for better resource utilization
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (; i < n; i += stride) {
        y[i] = a * x[i] + y[i];
    }
}