#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel0(int n, float a, float *x, float *y){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize x and y arrays outside the loop
    if (i < n) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Perform the computation
    if (i < n) {
        y[i] = a * x[i] + y[i];
    }
}