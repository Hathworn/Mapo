#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel0(int n, float a, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        y[i] = fmaf(a, x[i], y[i]);  // Use fmaf for better performance and precision
    }
}