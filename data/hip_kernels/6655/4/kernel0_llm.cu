#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel0(int n, float a, float *x, float *y) {
    // Calculate the global index of the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index does not exceed boundaries
    if (i < n) {
        // Initialize x and y values at index i
        x[i] = 1.0f;
        y[i] = 2.0f;

        // Compute the result for y[i]
        y[i] = a * x[i] + y[i];
    }
}