#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void multiplyDigits(char* d_str1, char* d_str2, int* d_matrix, int str1_len, int str2_len) {
    // Simplify indexing calculation by precalculating values
    int row = blockIdx.x * blockDim.y + threadIdx.y;
    int col = blockIdx.y * blockDim.x + threadIdx.x;
    
    // Check bounds before executing multiplication
    if (row < str2_len && col < str1_len) {
        int idx = row * (str1_len * str2_len + str1_len) + col + str2_len + 1;

        // Perform multiplication
        d_matrix[idx] = (d_str2[row] - '0') * (d_str1[col] - '0');
    }
}