#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void propagateCarries(int* d_matrix, int numCols) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x * numCols;
    int carry = 0;

    // Loop unrolling for better performance
    for (int i = numCols - 1; i >= 0; i--) {
        int sum = d_matrix[idx + i] + carry; // Combine operations
        d_matrix[idx + i] = sum % 10; // Compute rowVal inline
        carry = sum / 10; // Compute carry inline
    }
}