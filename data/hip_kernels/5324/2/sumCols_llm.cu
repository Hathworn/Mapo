#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void sumCols(int* d_matrix, int* d_result, int numRows, int numCols) {
    // Calculate unique column index for each thread
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure thread index is within valid range
    if (idx < numCols) {
        int sum = 0;

        // Accumulate column sums
        for (int i = 0; i < numRows; i++) {
            sum += d_matrix[idx + (numCols * i)];
        }

        // Store computed sum in the result array
        d_result[idx] = sum;
    }
}