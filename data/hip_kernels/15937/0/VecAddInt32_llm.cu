#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAddInt32(int32_t* in0, int32_t* in1, int32_t* out, int cnt)
{
    // Use shared memory for potential optimization
    extern __shared__ int32_t s_in0[];
    extern __shared__ int32_t s_in1[];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    // Load data into shared memory
    if (tid < cnt) {
        s_in0[local_tid] = in0[tid];
        s_in1[local_tid] = in1[tid];
        __syncthreads(); // Ensure all data is loaded before proceeding

        // Perform vector addition
        out[tid] = s_in0[local_tid] + s_in1[local_tid];
    }
}