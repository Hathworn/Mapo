#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAddFp32(float* in0, float* in1, float* out, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for grid-stride loop
    for (int i = tid; i < cnt; i += stride) { // Apply grid-stride loop
        out[i] = in0[i] + in1[i];
    }
}