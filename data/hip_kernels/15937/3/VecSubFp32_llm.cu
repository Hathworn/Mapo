#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecSubFp32(float* __restrict__ in0, float* __restrict__ in1, float* __restrict__ out, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;

    // Loop over the input arrays with a stride equal to the grid size
    for (int i = tid; i < cnt; i += gridSize) {
        out[i] = in0[i] - in1[i];
    }
}