#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecSubInt32(int32_t* in0, int32_t* in1, int32_t* out, int cnt)
{
    // Use shared memory for faster access if needed (uncomment if applicable)
    // __shared__ int32_t shared_in0[BLOCK_SIZE];
    // __shared__ int32_t shared_in1[BLOCK_SIZE];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Unrolling loop if there were multiple operations - Not applicable here, single operation
    if (tid < cnt) {
        out[tid] = in0[tid] - in1[tid];
    }
}