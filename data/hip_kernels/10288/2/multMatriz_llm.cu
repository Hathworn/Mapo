#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multMatriz(float *da, float *db, float *dc, int num) {
    float sum = 0;
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;

    // Avoid unnecessary loops by directly checking thread bounds
    if (i < num && j < num) {
        for (unsigned int k = 0; k < num; k++)
            sum += da[i * num + k] * db[k * num + j];
        dc[i * num + j] = sum;
    }
}