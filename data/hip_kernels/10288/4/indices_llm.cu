#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void indices(){
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Removed __syncthreads(), unnecessary since no shared data
    printf("blockdimy: %d  threadx: %d  Blockidx: %d  blockdimx: %d id:  %d raiz: %f\n",
           blockDim.y, threadIdx.x, blockIdx.x, blockDim.x, id, sqrt((double)id));
}