#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void suma(int a, int b, int *c) {
    // Use shared memory to minimize global memory access
    __shared__ int temp;
    if (threadIdx.x == 0) { // Ensure only one thread performs the calculation
        temp = a + b;
    }
    __syncthreads(); // Synchronize before reading from shared memory
    if (threadIdx.x == 0) {
        *c = temp;
    }
}