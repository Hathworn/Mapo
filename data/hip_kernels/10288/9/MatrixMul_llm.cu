#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 16 // Define the size of the tile

__global__ void MatrixMul(float *Md, float *Nd, float *Pd, const int WIDTH) {
    // Allocate shared memory for tiles
    __shared__ float Mds[TILE_SIZE][TILE_SIZE];
    __shared__ float Nds[TILE_SIZE][TILE_SIZE];

    // Calculate row and column index for matrix P
    unsigned int col = TILE_SIZE * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_SIZE * blockIdx.y + threadIdx.y;

    float Pvalue = 0; // Initialize Pvalue for the current element

    // Loop over tiles
    for (int t = 0; t < (WIDTH + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Collaborative loading of Md and Nd tiles into shared memory
        if (t * TILE_SIZE + threadIdx.x < WIDTH && row < WIDTH) {
            Mds[threadIdx.y][threadIdx.x] = Md[row * WIDTH + t * TILE_SIZE + threadIdx.x];
        } else {
            Mds[threadIdx.y][threadIdx.x] = 0.0;
        }
      
        if (t * TILE_SIZE + threadIdx.y < WIDTH && col < WIDTH) {
            Nds[threadIdx.y][threadIdx.x] = Nd[(t * TILE_SIZE + threadIdx.y) * WIDTH + col];
        } else {
            Nds[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads(); // Ensure matrices are loaded

        // Multiply tiles
        for (int k = 0; k < TILE_SIZE; k++) {
            Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
        }

        __syncthreads(); // Synchronize threads
    }

    // Write the result to the output matrix
    if (row < WIDTH && col < WIDTH) {
        Pd[row * WIDTH + col] = Pvalue;
    }
}