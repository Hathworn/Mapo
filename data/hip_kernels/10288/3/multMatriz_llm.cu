#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multMatriz(float *da, float *db, float *dc, int num) {
    float sum = 0;
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;

    // Loop over rows
    for (int row = i; row < num; row += gridDim.y * blockDim.y) {
        // Loop over columns
        for (int col = j; col < num; col += gridDim.x * blockDim.x) {
            sum = 0; // Reset sum for each element
            // Compute dot product
            for (unsigned int k = 0; k < num; k++) {
                sum += da[row * num + k] * db[k * num + col];
            }
            dc[row * num + col] = sum;
        }
    }
}