#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpy(double *y, double *a, double *x, int m, int n) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];
    
    // Use a thread block to handle multiple rows if possible
    int rowStep = gridDim.x;
    for (int row = bid; row < m; row += rowStep) {
        if (tid < n) {
            dots_s[tid] = a[row * n + tid] * x[tid]; // Store result in shared memory
        }
        __syncthreads(); // Synchronize to ensure all threads have written

        // Reduction within the block
        for (int stride = n / 2; stride > 0; stride >>= 1) {
            if (tid < stride && tid + stride < n) {
                dots_s[tid] += dots_s[tid + stride];
            }
            __syncthreads(); // Ensure stride reduction is complete before next stride
        }
        
        if (tid == 0) {
            y[row] = dots_s[0]; // First thread writes the result
        }
        __syncthreads();
    }
}