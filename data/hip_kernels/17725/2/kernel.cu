#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(hipSurfaceObject_t input, hipSurfaceObject_t output, int width, int height) {

//Get the pixel index
unsigned int xPx = threadIdx.x + blockIdx.x * blockDim.x;
unsigned int yPx = threadIdx.y + blockIdx.y * blockDim.y;


//Don't do any computation if this thread is outside of the surface bounds.
if (xPx >= width || yPx >= height) return;

//Copy the contents of input to output.
uchar4 pixel = {255,128,0,255};
//Read a pixel from the input. Disable to default to the flat orange color above
surf2Dread<uchar4>(&pixel, input, xPx * sizeof(uchar4), yPx, hipBoundaryModeClamp);
surf2Dwrite(pixel, output, xPx * sizeof(uchar4), yPx);
}