#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "TgvThresholdingL1MaskedKernel.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float2 *Tp = NULL;
hipMalloc(&Tp, XSIZE*YSIZE);
float *u_ = NULL;
hipMalloc(&u_, XSIZE*YSIZE);
float *Iu = NULL;
hipMalloc(&Iu, XSIZE*YSIZE);
float *Iz = NULL;
hipMalloc(&Iz, XSIZE*YSIZE);
float *mask = NULL;
hipMalloc(&mask, XSIZE*YSIZE);
float lambda = 1;
float tau = 1;
float *eta_u = NULL;
hipMalloc(&eta_u, XSIZE*YSIZE);
float *u = NULL;
hipMalloc(&u, XSIZE*YSIZE);
float *us = NULL;
hipMalloc(&us, XSIZE*YSIZE);
int width = XSIZE;
int height = YSIZE;
int stride = 2;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
TgvThresholdingL1MaskedKernel<<<gridBlock, threadBlock>>>(Tp,u_,Iu,Iz,mask,lambda,tau,eta_u,u,us,width,height,stride);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
TgvThresholdingL1MaskedKernel<<<gridBlock, threadBlock>>>(Tp,u_,Iu,Iz,mask,lambda,tau,eta_u,u,us,width,height,stride);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
TgvThresholdingL1MaskedKernel<<<gridBlock, threadBlock>>>(Tp,u_,Iu,Iz,mask,lambda,tau,eta_u,u,us,width,height,stride);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}