#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TgvThresholdingL1MaskedKernel(float2* Tp, float* u_, float* Iu, float* Iz, float* mask, float lambda, float tau, float* eta_u, float* u, float* us, int width, int height, int stride)
{
    int iy = blockIdx.y * blockDim.y + threadIdx.y;   // current row
    int ix = blockIdx.x * blockDim.x + threadIdx.x;   // current column

    if (iy >= height || ix >= width) return;          // bounds check optimization
    int pos = ix + iy * stride;
    if (mask[pos] == 0.0f) return;                    // skip computation if mask is zero

    int right = (ix + 1) + iy * stride;
    int down = ix + (iy + 1) * stride;
    int left = (ix - 1) + iy * stride;
    int up = ix + (iy - 1) * stride;

    float maskRight = (ix + 1 < width) ? mask[right] : 0.0f;
    float maskLeft = (ix - 1 >= 0) ? mask[left] : 0.0f;
    float maskDown = (iy + 1 < height) ? mask[down] : 0.0f;
    float maskUp = (iy - 1 >= 0) ? mask[up] : 0.0f;

    // compute divergence of p
    float dxmTp = (maskLeft != 0.0f && maskRight != 0.0f) ? (Tp[pos].x - Tp[left].x) : 
                 (maskRight == 0.0f) ? -Tp[left].x : Tp[pos].x;

    float dymTp = (maskUp != 0.0f && maskDown != 0.0f) ? (Tp[pos].y - Tp[up].y) : 
                 (maskDown == 0.0f) ? -Tp[up].y : Tp[pos].y;

    float div_p = dxmTp + dymTp;

    // compute modified tau value
    float tau_eta_u = (eta_u[pos] == 0) ? tau : (tau / eta_u[pos]);

    // Thresholding
    float uhat = u_[pos] + tau_eta_u * div_p;
    float dun = (uhat - u[pos]);
    float Ius = Iu[pos];
    float rho = Ius * dun + Iz[pos];

    float upper = lambda * tau_eta_u * (Ius * Ius);
    float lower = -upper;
    float du = dun;

    if ((rho > upper)) {
        du -= lambda * tau_eta_u * Ius;
    } else if (rho < lower) {
        du += lambda * tau_eta_u * Ius;
    } else if (Ius != 0) {
        du -= rho / Ius;
    }

    us[pos] = u[pos] + du;
}