#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TgvCloneKernel(float* dst, float* src, int width, int height, int stride) {
    // Calculate the global thread index
    int pos = blockIdx.y * blockDim.y * stride + blockIdx.x * blockDim.x + threadIdx.y * stride + threadIdx.x;
    
    // Check if within bounds
    if (pos < width * height) {
        dst[pos] = src[pos];
    }
}