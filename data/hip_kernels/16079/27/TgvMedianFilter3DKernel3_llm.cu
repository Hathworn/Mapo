#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TgvMedianFilter3DKernel3(float* X, float* Y, float *Z, int width, int height, int stride, float *X1, float *Y1, float *Z1)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;

    const int pos = ix + iy * stride;

    if (ix >= width || iy >= height) return;

    float mX[9], mY[9], mZ[9];

    // Store indices for boundary checks
    int col, row, index;
    int last_col = width - 1;
    int last_row = height - 1;

    // Load data using the 3x3 neighborhood
    for (int j = 0; j < 3; j++) {
        for (int i = 0; i < 3; i++) {
            col = min(max(ix + i - 1, 0), last_col);
            row = min(max(iy + j - 1, 0), last_row);
            index = j * 3 + i;
            mX[index] = X[col + stride * row];
            mY[index] = Y[col + stride * row];
            mZ[index] = Z[col + stride * row];
        }
    }

    // Perform in-place selection sort for median filter
    for (int j = 0; j < 5; j++) {
        for (int i = j + 1; i < 9; i++) {
            if (mX[j] > mX[i]) {
                // Swap the elements
                float tmp = mX[j];
                mX[j] = mX[i];
                mX[i] = tmp;
            }
            if (mY[j] > mY[i]) {
                // Swap the elements
                float tmp = mY[j];
                mY[j] = mY[i];
                mY[i] = tmp;
            }
            if (mZ[j] > mZ[i]) {
                // Swap the elements
                float tmp = mZ[j];
                mZ[j] = mZ[i];
                mZ[i] = tmp;
            }
        }
    }

    // Store the median value to output arrays
    X1[pos] = mX[4];
    Y1[pos] = mY[4];
    Z1[pos] = mZ[4];
}