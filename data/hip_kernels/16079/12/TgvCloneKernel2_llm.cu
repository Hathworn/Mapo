#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TgvCloneKernel2(float2* dst, float2* src, int width, int height, int stride) {
    int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row
    int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column

    if ((iy < height) && (ix < width)) {
        int pos = ix + iy * stride;
        // Unroll the store operation for potential performance gains
        float2 srcValue = src[pos];
        dst[pos] = srcValue;
    }
}