#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TgvComputeOpticalFlowVectorKernel(const float *u, const float2 *tv2, int width, int height, int stride, float2 *warpUV)
{
    // Calculate global thread indices
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds before calculating position
    if (ix < width && iy < height) 
    {
        const int pos = ix + iy * stride;
         
        // Load values from global memory
        float us = u[pos];
        float2 tv2s = tv2[pos];
        
        // Write results back to global memory
        warpUV[pos].x = us * tv2s.x;
        warpUV[pos].y = us * tv2s.y;
    }
}