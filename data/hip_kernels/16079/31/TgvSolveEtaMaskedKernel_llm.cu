#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TgvSolveEtaMaskedKernel(float* mask, float alpha0, float alpha1, float* atensor, float *btensor, float* ctensor, float* etau, float* etav1, float* etav2, int width, int height, int stride)
{
    int iy = blockIdx.y * blockDim.y + threadIdx.y;  // current row
    int ix = blockIdx.x * blockDim.x + threadIdx.x;  // current column

    // Ensure indices are within bounds
    if (iy >= height || ix >= width) return;

    int pos = ix + iy * stride;
    
    // Check mask condition early
    if (mask[pos] == 0.0f) return;

    float a = atensor[pos];
    float b = btensor[pos];
    float c = ctensor[pos];

    // Precompute reusable expressions
    float alpha1_sq = alpha1 * alpha1;
    float alpha0_sq_4 = 4 * alpha0 * alpha0;
    float a_plus_c = a + c;
    float b_plus_c = b + c;
    
    // Combined computation
    etau[pos] = (a * a + b * b + 2 * c * c + a_plus_c * a_plus_c + b_plus_c * b_plus_c) * alpha1_sq;
    etav1[pos] = (alpha1_sq * (b * b + c * c)) + alpha0_sq_4;
    etav2[pos] = (alpha1_sq * (a * a + c * c)) + alpha0_sq_4;
}