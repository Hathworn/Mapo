#include "hip/hip_runtime.h"
```cuda
#include "hip/hip_runtime.h"
#include "includes.h"

/// scalar field to upscale
texture<float, hipTextureType2D, hipReadModeElementType> texCoarse;
texture<float2, hipTextureType2D, hipReadModeElementType> texCoarseFloat2;

__global__
__global__ void TgvUpscaleMaskedKernel(float * mask, int width, int height, int stride, float scale, float *out)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure thread is within bounds
    if (ix >= width || iy >= height) return;

    int pos = ix + iy * stride;
    
    // Interpolate texture value and scale
    float x = ((float)ix + 0.5f) / (float)width;
    float y = ((float)iy + 0.5f) / (float)height;
    out[pos] = tex2D(texCoarse, x, y) * scale;
}