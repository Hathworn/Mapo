#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TgvMedianFilterKernel3(float* u, float* v, int width, int height, int stride, float *outputu, float *outputv)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    const int pos = ix + iy * stride;

    if (ix >= width || iy >= height) return;

    float mu[9], mv[9];

    #pragma unroll
    for (int j = 0; j < 3; j++) {
        #pragma unroll
        for (int i = 0; i < 3; i++) {
            int col = ix + i - 1;
            int row = iy + j - 1;
            int index = j * 3 + i;
            
            // Load boundary conditions
            if ((col >= 0) && (col < width) && (row >= 0) && (row < height)) {
                mu[index] = u[col + stride * row];
                mv[index] = v[col + stride * row];
            } else {
                mu[index] = (col < 0) ? u[stride * row] : (col >= width) ? u[width - 1 + stride * row] : (row < 0) ? u[col] : u[col + stride * (height - 1)];
                mv[index] = (col < 0) ? v[stride * row] : (col >= width) ? v[width - 1 + stride * row] : (row < 0) ? v[col] : v[col + stride * (height - 1)];
            }
        }
    }

    // Insertion sort to find median
    #pragma unroll
    for (int j = 0; j < 9; j++) {
        #pragma unroll
        for (int i = j + 1; i < 9; i++) {
            if (mu[j] > mu[i]) {
                float tmp = mu[j];
                mu[j] = mu[i];
                mu[i] = tmp;
            }
            if (mv[j] > mv[i]) {
                float tmp = mv[j];
                mv[j] = mv[i];
                mv[i] = tmp;
            }
        }
    }

    outputu[pos] = mu[4];
    outputv[pos] = mv[4];
}