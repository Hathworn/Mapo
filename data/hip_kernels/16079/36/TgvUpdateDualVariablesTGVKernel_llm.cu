#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TgvUpdateDualVariablesTGVKernel(float* u_, float2 *v_, float alpha0, float alpha1, float sigma, float eta_p, float eta_q, float* a, float* b, float* c, float4* grad_v, float2* p, float4* q, int width, int height, int stride)
{
    int iy = blockIdx.y * blockDim.y + threadIdx.y; // current row
    int ix = blockIdx.x * blockDim.x + threadIdx.x; // current column

    float desiredRadius = (float)width / 2.20f;
    float halfWidth = (float)width / 2.0f;
    float halfHeight = (float)height / 2.0f;
    float radius = sqrtf((iy - halfHeight) * (iy - halfHeight) + (ix - halfWidth) * (ix - halfWidth));

    if ((iy < height) && (ix < width))
    {
        int pos = ix + iy * stride;

        if (radius >= desiredRadius)
        {
            p[pos] = make_float2(0.0f, 0.0f);
            q[pos] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        }
        else
        {
            // Pre-calculate multipliers to avoid repeated computation
            float alpha1_sigma_eta_p = alpha1 * sigma / eta_p;
            float alpha0_sigma_eta_q = alpha0 * sigma / eta_q;

            // Calculate boundary conditions safely
            int right = min(ix + 1, width - 1) + iy * stride;
            int down = ix + min(iy + 1, height - 1) * stride;
            int left = max(ix - 1, 0) + iy * stride;
            int up = ix + max(iy - 1, 0) * stride;

            // u_x and u_y calculations
            float u_x = u_[right] - u_[pos] - v_[pos].x;
            float u_y = u_[down] - u_[pos] - v_[pos].y;

            // du_tensor calculations
            float du_tensor_x = a[pos] * u_x + c[pos] * u_y;
            float du_tensor_y = c[pos] * u_x + b[pos] * u_y;

            // Update p
            p[pos].x += alpha1_sigma_eta_p * du_tensor_x;
            p[pos].y += alpha1_sigma_eta_p * du_tensor_y;

            // Projection for p
            float reprojection = fmaxf(1.0f, sqrtf(p[pos].x * p[pos].x + p[pos].y * p[pos].y));
            p[pos] /= reprojection;

            // grad_v calculations
            grad_v[pos].x = v_[right].x - v_[pos].x;
            grad_v[pos].y = v_[down].y - v_[pos].y;
            grad_v[pos].z = v_[down].x - v_[pos].x;
            grad_v[pos].w = v_[right].y - v_[pos].y;

            // Update q
            float4 gv = grad_v[pos];
            q[pos].x += alpha0_sigma_eta_q * gv.x;
            q[pos].y += alpha0_sigma_eta_q * gv.y;
            q[pos].z += alpha0_sigma_eta_q * gv.z;
            q[pos].w += alpha0_sigma_eta_q * gv.w;

            // Projection for q
            float reproject = fmaxf(1.0f, sqrtf(q[pos].x * q[pos].x + q[pos].y * q[pos].y + q[pos].z * q[pos].z + q[pos].w * q[pos].w));
            q[pos] /= reproject;
        }
    }
}