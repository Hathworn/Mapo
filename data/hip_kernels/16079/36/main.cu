#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "TgvUpdateDualVariablesTGVKernel.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *u_ = NULL;
hipMalloc(&u_, XSIZE*YSIZE);
float2 *v_ = NULL;
hipMalloc(&v_, XSIZE*YSIZE);
float alpha0 = 2;
float alpha1 = 2;
float sigma = 1;
float eta_p = 1;
float eta_q = 1;
float *a = NULL;
hipMalloc(&a, XSIZE*YSIZE);
float *b = NULL;
hipMalloc(&b, XSIZE*YSIZE);
float *c = NULL;
hipMalloc(&c, XSIZE*YSIZE);
float4 *grad_v = NULL;
hipMalloc(&grad_v, XSIZE*YSIZE);
float2 *p = NULL;
hipMalloc(&p, XSIZE*YSIZE);
float4 *q = NULL;
hipMalloc(&q, XSIZE*YSIZE);
int width = XSIZE;
int height = YSIZE;
int stride = 2;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
TgvUpdateDualVariablesTGVKernel<<<gridBlock, threadBlock>>>(u_,v_,alpha0,alpha1,sigma,eta_p,eta_q,a,b,c,grad_v,p,q,width,height,stride);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
TgvUpdateDualVariablesTGVKernel<<<gridBlock, threadBlock>>>(u_,v_,alpha0,alpha1,sigma,eta_p,eta_q,a,b,c,grad_v,p,q,width,height,stride);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
TgvUpdateDualVariablesTGVKernel<<<gridBlock, threadBlock>>>(u_,v_,alpha0,alpha1,sigma,eta_p,eta_q,a,b,c,grad_v,p,q,width,height,stride);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}