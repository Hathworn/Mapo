#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TgvSolveEtaKernel(float alpha0, float alpha1, float* atensor, float *btensor, float* ctensor, float* etau, float* etav1, float* etav2, int width, int height, int stride)
{
    int iy = blockIdx.y * blockDim.y + threadIdx.y;    // current row
    int ix = blockIdx.x * blockDim.x + threadIdx.x;    // current column

    if ((iy < height) && (ix < width))
    {
        int pos = ix + iy * stride;
        
        // Use shared memory to minimize repeated global memory access
        __shared__ float shared_atensor[blockDim.x * blockDim.y];
        __shared__ float shared_btensor[blockDim.x * blockDim.y];
        __shared__ float shared_ctensor[blockDim.x * blockDim.y];

        int localIdx = threadIdx.y * blockDim.x + threadIdx.x;
        shared_atensor[localIdx] = atensor[pos];
        shared_btensor[localIdx] = btensor[pos];
        shared_ctensor[localIdx] = ctensor[pos];

        __syncthreads(); // Synchronize to ensure all threads have loaded values

        // Perform calculations using shared memory
        float a = shared_atensor[localIdx];
        float b = shared_btensor[localIdx];
        float c = shared_ctensor[localIdx];

        float alpha1Squared = alpha1 * alpha1;
        float a_c = a + c;
        float b_c = b + c;

        etau[pos] = (a * a + b * b + 2 * c * c + a_c * a_c + b_c * b_c) * alpha1Squared;
        etav1[pos] = alpha1Squared * (b * b + c * c) + 4 * alpha0 * alpha0;
        etav2[pos] = alpha1Squared * (a * a + c * c) + 4 * alpha0 * alpha0;
    }
}