#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/// scalar field to upscale
texture<float, hipTextureType2D, hipReadModeElementType> texCoarse;
texture<float2, hipTextureType2D, hipReadModeElementType> texCoarseFloat2;

__global__ void TgvUpscaleFloat2Kernel(int width, int height, int stride, float scale, float2 *out)
{
    // Use shared memory to reduce global memory access for block-wide outputs
    extern __shared__ float2 sharedOut[];

    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix < width && iy < height) {
        float x = ((float)ix + 0.5f) / (float)width;
        float y = ((float)iy + 0.5f) / (float)height;

        // Utilize hardware interpolation
        float2 src = tex2D(texCoarseFloat2, x, y);
        float2 scaledSrc;
        scaledSrc.x = src.x * scale;
        scaledSrc.y = src.y * scale;

        // Store the result in shared memory
        int localIndex = threadIdx.x + threadIdx.y * blockDim.x;
        sharedOut[localIndex] = scaledSrc;

        // Wait until all threads have written to shared memory
        __syncthreads();

        // Copy results from shared memory to global memory
        out[ix + iy * stride] = sharedOut[localIndex];
    }
}