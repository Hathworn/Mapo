#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TgvUpdateDualVariablesTGVMaskedKernel(float* mask, float* u_, float2 *v_, float alpha0, float alpha1, float sigma, float eta_p, float eta_q, float* a, float* b, float*c, float4* grad_v, float2* p, float4* q, int width, int height, int stride)
{
    int iy = blockIdx.y * blockDim.y + threadIdx.y;        // current row
    int ix = blockIdx.x * blockDim.x + threadIdx.x;        // current column

    if (iy >= height || ix >= width) return;               // Efficient boundary check
    int pos = ix + iy * stride;
    if (mask[pos] == 0.0f) return;                         // Skip masked elements

    int right = (ix + 1) + iy * stride;
    int down = ix + (iy + 1) * stride;
    int left = (ix - 1) + iy * stride;
    int up = ix + (iy - 1) * stride;
    
    float maskRight = (ix + 1 >= width) ? 0.0f : mask[right];
    float maskDown = (iy + 1 >= height) ? 0.0f : mask[down];

    float u_pos = u_[pos];
    float2 v_pos = v_[pos];

    float u_x = (maskRight != 0.0f) ? (u_[right] - u_pos - v_pos.x) : (u_pos - u_[left] - v_pos.x);
    float u_y = (maskDown != 0.0f) ? (u_[down] - u_pos - v_pos.y) : (u_pos - u_[up] - v_pos.y);

    float du_tensor_x = a[pos] * u_x + c[pos] * u_y;
    float du_tensor_y = c[pos] * u_x + b[pos] * u_y;

    float2 ppos;
    float inv_eta_p = alpha1 * sigma / eta_p;              // Precompute division
    ppos.x = p[pos].x + inv_eta_p * du_tensor_x;
    ppos.y = p[pos].y + inv_eta_p * du_tensor_y;

    float reprojection = fmaxf(1.0f, sqrtf(ppos.x * ppos.x + ppos.y * ppos.y));
    p[pos].x = ppos.x / reprojection;
    p[pos].y = ppos.y / reprojection;

    float4 grad_v_pos;
    grad_v_pos.x = (maskRight != 0.0f) ? (v_[right].x - v_pos.x) : (v_pos.x - v_[left].x);
    grad_v_pos.y = (maskDown != 0.0f) ? (v_[down].y - v_pos.y) : (v_pos.y - v_[up].y);
    grad_v_pos.z = (maskDown != 0.0f) ? (v_[down].x - v_pos.x) : (v_pos.x - v_[up].x);
    grad_v_pos.w = (maskRight != 0.0f) ? (v_[right].y - v_pos.y) : (v_pos.y - v_[left].y);

    grad_v[pos] = grad_v_pos;

    float ase = alpha0 * sigma / eta_q;                    // Precompute division
    float4 qpos;
    qpos.x = q[pos].x + ase * grad_v_pos.x;
    qpos.y = q[pos].y + ase * grad_v_pos.y;
    qpos.z = q[pos].z + ase * grad_v_pos.z;
    qpos.w = q[pos].w + ase * grad_v_pos.w;

    float reproject = fmaxf(1.0f, sqrtf(qpos.x * qpos.x + qpos.y * qpos.y + qpos.z * qpos.z + qpos.w * qpos.w));
    q[pos].x = qpos.x / reproject;
    q[pos].y = qpos.y / reproject;
    q[pos].z = qpos.z / reproject;
    q[pos].w = qpos.w / reproject;
}