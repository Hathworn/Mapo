#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TgvThresholdingL1Kernel(float2* Tp, float* u_, float* Iu, float* Iz, float lambda, float tau, float* eta_u, float* u, float* us, int width, int height, int stride)
{
    int iy = blockIdx.y * blockDim.y + threadIdx.y; // current row
    int ix = blockIdx.x * blockDim.x + threadIdx.x; // current column

    float desiredRadius = (float)width / 2.20f;
    float halfWidth = (float)width / 2.0f;
    float halfHeight = (float)height / 2.0f;
    float radius = sqrtf((iy - halfHeight) * (iy - halfHeight) + (ix - halfWidth) * (ix - halfWidth));

    if ((iy < height) && (ix < width))
    {
        int pos = ix + iy * stride;
        if (radius >= desiredRadius)
        {
            us[pos] = 0.0f;
        }
        else
        {
            int left = (ix - 1) + iy * stride;
            int up = ix + (iy - 1) * stride;

            // Calculate divergence of p using conditional assignments
            float dxmTp = (ix > 0) ? (Tp[pos].x - Tp[left].x) : Tp[pos].x;
            float dymTp = (iy > 0) ? (Tp[pos].y - Tp[up].y) : Tp[pos].y;
            float div_p = dxmTp + dymTp;

            // Calculate tau_eta_u using ternary operator
            float tau_eta_u = (eta_u[pos] == 0) ? tau : (tau / eta_u[pos]);

            // Thresholding
            float uhat = u_[pos] + tau_eta_u * div_p;
            float dun = (uhat - u[pos]);
            float Ius = Iu[pos];
            float rho = Ius * dun + Iz[pos];
            float scaledLambdaTau = lambda * tau_eta_u * (Ius * Ius);
            
            // Simplify conditionals using inequalities
            float du = dun;
            if (rho < -scaledLambdaTau)
            {
                du += lambda * tau_eta_u * Ius;
            }
            else if (rho > scaledLambdaTau)
            {
                du -= lambda * tau_eta_u * Ius;
            }
            else if (Ius != 0)
            {
                du -= rho / Ius;
            }

            us[pos] = u[pos] + du;
        }
    }
}