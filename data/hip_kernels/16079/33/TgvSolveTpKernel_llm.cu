#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TgvSolveTpKernel(float *a, float *b, float *c, float2 *p, float2 *Tp, int width, int height, int stride) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;     // current column
    int iy = blockIdx.y * blockDim.y + threadIdx.y;     // current row
    
    if ((ix < width) && (iy < height)) {                // ensure within bounds
        int pos = ix + iy * stride;

        float px = p[pos].x;
        float py = p[pos].y;
        // Reduce memory access by storing values into registers
        Tp[pos].x = a[pos] * px + c[pos] * py;
        Tp[pos].y = c[pos] * px + b[pos] * py;
    }
}