#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TgvSolveTpMaskedKernel(float* mask, float*a, float *b, float*c, float2* p, float2* Tp, int width, int height, int stride) {
    // Calculate 1D flattened thread index
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check thread bounds 
    if (ix >= width || iy >= height) return;

    int pos = ix + iy * stride;

    // Check mask condition
    if (mask[pos] == 0.0f) return;

    // Perform computations
    float2 pVal = p[pos];
    float2 tpVal;
    tpVal.x = a[pos] * pVal.x + c[pos] * pVal.y;
    tpVal.y = c[pos] * pVal.x + b[pos] * pVal.y;
    Tp[pos] = tpVal;
}