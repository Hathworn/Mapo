#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "TgvComputeOpticalFlowVectorMaskedKernel.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    const float *u = NULL;
hipMalloc(&u, XSIZE*YSIZE);
const float2 *tv2 = NULL;
hipMalloc(&tv2, XSIZE*YSIZE);
float *mask = NULL;
hipMalloc(&mask, XSIZE*YSIZE);
int width = XSIZE;
int height = YSIZE;
int stride = 2;
float2 *warpUV = NULL;
hipMalloc(&warpUV, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
TgvComputeOpticalFlowVectorMaskedKernel<<<gridBlock, threadBlock>>>(u,tv2,mask,width,height,stride,warpUV);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
TgvComputeOpticalFlowVectorMaskedKernel<<<gridBlock, threadBlock>>>(u,tv2,mask,width,height,stride,warpUV);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
TgvComputeOpticalFlowVectorMaskedKernel<<<gridBlock, threadBlock>>>(u,tv2,mask,width,height,stride,warpUV);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}