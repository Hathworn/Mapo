#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TgvComputeOpticalFlowVectorMaskedKernel(const float *u, const float2 *tv2, float* mask, int width, int height, int stride, float2 *warpUV)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure threads are within bounds
    if (iy < height && ix < width) {
        int pos = ix + iy * stride;
        // Only process if mask allows
        if (mask[pos] != 0.0f) {
            float us = u[pos];
            float2 tv2s = tv2[pos];
            warpUV[pos].x = us * tv2s.x;
            warpUV[pos].y = us * tv2s.y;
        }
    }
}