#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "TgvConvertKBKernel.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float2 *disparity = NULL;
hipMalloc(&disparity, XSIZE*YSIZE);
float focalx = 1;
float focaly = 1;
float cx = 1;
float cy = 1;
float d1 = 2;
float d2 = 2;
float d3 = 2;
float d4 = 1;
float t1 = 1;
float t2 = 1;
float t3 = 1;
float3 *X = NULL;
hipMalloc(&X, XSIZE*YSIZE);
float *depth = NULL;
hipMalloc(&depth, XSIZE*YSIZE);
int width = XSIZE;
int height = YSIZE;
int stride = 2;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
TgvConvertKBKernel<<<gridBlock, threadBlock>>>(disparity,focalx,focaly,cx,cy,d1,d2,d3,d4,t1,t2,t3,X,depth,width,height,stride);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
TgvConvertKBKernel<<<gridBlock, threadBlock>>>(disparity,focalx,focaly,cx,cy,d1,d2,d3,d4,t1,t2,t3,X,depth,width,height,stride);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
TgvConvertKBKernel<<<gridBlock, threadBlock>>>(disparity,focalx,focaly,cx,cy,d1,d2,d3,d4,t1,t2,t3,X,depth,width,height,stride);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}