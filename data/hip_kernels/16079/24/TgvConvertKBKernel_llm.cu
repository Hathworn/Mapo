#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TgvConvertKBKernel(float2* disparity, float focalx, float focaly, float cx, float cy, float d1, float d2, float d3, float d4, float t1, float t2, float t3, float3* X, float* depth, int width, int height, int stride)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;

    const int pos = ix + iy * stride;

    if (ix >= width || iy >= height) return;

    float u0 = (float)ix;
    float v0 = (float)iy;
    float xprime0 = (u0 - focalx) / cx;
    float yprime0 = (v0 - focaly) / cy;

    float u = disparity[pos].x;
    float v = disparity[pos].y;

    float u1 = u0 + u;
    float v1 = v0 + v;
    float xprime1 = (u1 - focalx) / cx;
    float yprime1 = (v1 - focaly) / cy;

    // Newton-Raphson Method Frame 0
    float ru0 = sqrtf(xprime0 * xprime0 + yprime0 * yprime0);
    float theta0 = 0.0f;

    // Precompute powers of theta0 to avoid repeated calculations
    float theta0_2 = theta0 * theta0;
    float theta0_4 = theta0_2 * theta0_2;
    float theta0_6 = theta0_4 * theta0_2;
    float theta0_8 = theta0_4 * theta0_4;

    for (int iter = 0; iter < 5; iter++) {
        float thetad0 = theta0 + d1 * theta0_2 * theta0 + d2 * theta0_4 * theta0 + d3 * theta0_6 * theta0 + d4 * theta0_8 * theta0;
        float Dthetad0 = 1.0f + 3.0f * d1 * theta0_2 + 5.0f * d2 * theta0_4 + 7.0f * d3 * theta0_6 + 9.0f * d4 * theta0_8;
        float f0 = ru0 - thetad0;
        float f0prime = -Dthetad0;
        theta0 -= f0 / f0prime;
        
        // Update precomputed values
        theta0_2 = theta0 * theta0;
        theta0_4 = theta0_2 * theta0_2;
        theta0_6 = theta0_4 * theta0_2;
        theta0_8 = theta0_4 * theta0_4;
    }
    
    float x0out = tanf(theta0) * xprime0 / ru0;
    float y0out = tanf(theta0) * yprime0 / ru0;

    // Newton-Raphson Method Frame 1
    float ru1 = sqrtf(xprime1 * xprime1 + yprime1 * yprime1);
    float theta1 = 0.0f;

    // Precompute powers of theta1 to avoid repeated calculations
    float theta1_2 = theta1 * theta1;
    float theta1_4 = theta1_2 * theta1_2;
    float theta1_6 = theta1_4 * theta1_2;
    float theta1_8 = theta1_4 * theta1_4;

    for (int iter = 0; iter < 5; iter++) {
        float thetad1 = theta1 + d1 * theta1_2 * theta1 + d2 * theta1_4 * theta1 + d3 * theta1_6 * theta1 + d4 * theta1_8 * theta1;
        float Dthetad1 = 1.0f + 3.0f * d1 * theta1_2 + 5.0f * d2 * theta1_4 + 7.0f * d3 * theta1_6 + 9.0f * d4 * theta1_8;
        float f1 = ru1 - thetad1;
        float f1prime = -Dthetad1;
        theta1 -= f1 / f1prime;

        // Update precomputed values
        theta1_2 = theta1 * theta1;
        theta1_4 = theta1_2 * theta1_2;
        theta1_6 = theta1_4 * theta1_2;
        theta1_8 = theta1_4 * theta1_4;
    }
    
    float x1out = tanf(theta1) * xprime1 / ru1;
    float y1out = tanf(theta1) * yprime1 / ru1;

    // Triangulation
    float Zx = (t1 - x1out * t3) / (x1out - x0out);
    float Zy = (t2 - y1out * t3) / (y1out - y0out);
    float Z = Zx;

    X[pos].x = x0out * Z;
    X[pos].y = y0out * Z;
    X[pos].z = Z;
    
    depth[pos] = sqrt(X[pos].x * X[pos].x + X[pos].y * X[pos].y + X[pos].z * X[pos].z);
}