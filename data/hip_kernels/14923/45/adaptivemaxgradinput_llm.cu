#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w)
{
    // Calculate global thread position
    int o = blockIdx.x;
    int xx = threadIdx.x + blockIdx.y * blockDim.x;
    int yy = blockIdx.y * blockDim.y + threadIdx.y;

    // Return if the thread is outside the output dimensions
    if (xx >= output_w || yy >= output_h) return;
    
    // Offsets into respective arrays for the current batch
    gradOutput += o * output_w * output_h;
    gradInput += o * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;
    
    // Compute y_start and x_start only once per thread
    int x_start = __float2int_rd(static_cast<float>(xx) / output_w * input_w);
    int y_start = __float2int_rd(static_cast<float>(yy) / output_h * input_h);

    // Pointers for accessing specific positions in the arrays
    float *ptr_gradInput = gradInput + y_start * input_w + x_start;
    float *ptr_gradOutput = gradOutput + yy * output_w + xx;
    float *ptr_ind_x = indices_x + yy * output_w + xx;
    float *ptr_ind_y = indices_y + yy * output_w + xx;
    float z = *ptr_gradOutput;

    // Compute argmax positions
    int argmax_x = static_cast<int>(*ptr_ind_x) - 1;
    int argmax_y = static_cast<int>(*ptr_ind_y) - 1;

    // Accumulate gradient input
    ptr_gradInput[argmax_x + argmax_y * input_w] += z;
}