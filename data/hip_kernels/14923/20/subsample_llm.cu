#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output size only once
    const int output_w = (input_w - kW) / dW + 1;
    const int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    const int o = blockIdx.x;
    const int i = o;
    const int k = blockIdx.x % input_n;

    const int xx_start = threadIdx.x;
    const int xx_step = blockDim.x;

    const int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    const int yy_step = blockDim.y * gridDim.y;

    // Offset output and input pointers once
    float* out_ptr = output + o * output_w * output_h;
    float* in_ptr = input + i * input_w * input_h;

    // Get the weight and bias for this block
    const float the_weight = weight[k];
    const float the_bias = bias[k];

    // Iterate over output pixels
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            // Initialize sum for current output element
            float sum = 0;

            // Iterate over the kernel size
            for (int ky = 0; ky < kH; ky++) {
                // Calculate input row start position
                float* row_ptr = in_ptr + (yy * dH + ky) * input_w + xx * dW;
                
                // Accumulate sum for current (yy, xx) output position
                for (int kx = 0; kx < kW; kx++) {
                    sum += row_ptr[kx];
                }
            }
            // Update the output with weighted sum and bias
            out_ptr[yy * output_w + xx] = the_weight * sum + the_bias;
        }
    }
}
```
