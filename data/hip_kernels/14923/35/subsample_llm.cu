#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subsample(float *input, float *output, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int xx = threadIdx.x;
    int yy = blockDim.y * blockIdx.y + threadIdx.y;

    // Boundary check for y-dimension
    if (yy >= output_h) return;
    
    // Select input/output plane
    output += o * output_w * output_h;
    input += o * input_w * input_h;

    // For all output pixels, process in x-dimension (optimized loop)
    for (; xx < output_w; xx += blockDim.x) {
        // Compute the mean of the input image
        float *ptr_input = input + yy * dH * input_w + xx * dW;
        float sum = 0;
        for (int ky = 0; ky < kH; ky++) {
            for (int kx = 0; kx < kW; kx++) {
                sum += ptr_input[kx];
            }
            ptr_input += input_w; // Move to the next input line
        }
        // Update output
        output[yy * output_w + xx] = sum / float(kW * kH);
    }
}