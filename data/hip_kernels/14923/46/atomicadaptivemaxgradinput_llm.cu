#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void atomicadaptivemaxgradinput( float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w )
{
    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx_start = threadIdx.x + blockIdx.y * blockDim.x; // Increase parallelism
    int yy_start = threadIdx.y + blockIdx.z * blockDim.y;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // Compute gradInput
    for (int yy = yy_start; yy < output_h; yy += gridDim.y * blockDim.y) {
        int y_start = static_cast<int>(float(yy) / output_h * input_h);

        for (int xx = xx_start; xx < output_w; xx += gridDim.x * blockDim.x) {
            int x_start = static_cast<int>(float(xx) / output_w * input_w);

            float *ptr_gradInput = gradInput + y_start * input_w + x_start;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float *ptr_ind_x = indices_x + yy * output_w + xx;
            float *ptr_ind_y = indices_y + yy * output_w + xx;
            float z = *ptr_gradOutput;

            int argmax_x = static_cast<int>(*ptr_ind_x) - 1;
            int argmax_y = static_cast<int>(*ptr_ind_y) - 1;

            // Atomic add since different threads could update the same variable
            atomicAdd(&ptr_gradInput[argmax_x + argmax_y * input_w], z);
        }
    }
}