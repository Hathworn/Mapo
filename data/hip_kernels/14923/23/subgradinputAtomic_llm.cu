#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Compute grid and block indices for current thread
    int xx = threadIdx.x + blockIdx.y * blockDim.x;
    int yy = threadIdx.y + blockIdx.z * blockDim.y;

    // Output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    if (xx < output_w && yy < output_h) {
        // Select current plane
        int o = blockIdx.x;
        int i = o;
        int k = blockIdx.x % input_n;

        // Calculate data offsets
        gradOutput = gradOutput + o * output_w * output_h;
        gradInput = gradInput + i * input_w * input_h;

        // Get weight for the current thread
        float the_weight = weight[k];

        // Compute gradInput
        float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float z = *ptr_gradOutput * the_weight;

        // Accumulate using atomic operations
        for (int ky = 0; ky < kH; ky++) {
            for (int kx = 0; kx < kW; kx++) {
                atomicAdd(&(ptr_gradInput[kx]), z);
            }
            ptr_gradInput += input_w;
        }
    }
}