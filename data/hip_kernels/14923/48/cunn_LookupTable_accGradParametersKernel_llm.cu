#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_LookupTable_accGradParametersKernel(float *input, float *indices, float *gradOutput, float *gradWeight, float *count, float defaultScale, long numel, long stride) {

    int idx = blockIdx.x * 4 + threadIdx.y;

    // Optimized: Ensure idx is valid, and previous input is different or idx is 0
    if (idx < numel && (idx == 0 || input[idx] != input[idx - 1])) {
        // Optimized: Use a while loop efficiently to process repeated inputs
        do {
            const int startFeature = threadIdx.x + blockIdx.y * blockDim.x * 4;
            const int weightRow = __float2int_rd(input[idx]) * stride - stride; // simplified calculation
            const int gradOutputRow = __float2int_rd(indices[idx]) * stride - stride; // simplified calculation
            const float scale = count ? defaultScale / count[idx] : defaultScale;

            float gradient[4];
            float weight[4];

            #pragma unroll
            for (int ii = 0; ii < 4; ii++) {
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride) {
                    int gradOutputIndex = gradOutputRow + featureDim;
                    int gradWeightIndex = weightRow + featureDim;
                    gradient[ii] = gradOutput[gradOutputIndex];
                    weight[ii] = gradWeight[gradWeightIndex];
                }
            }

            #pragma unroll
            for (int ii = 0; ii < 4; ii++) {
                weight[ii] += gradient[ii] * scale;
            }

            #pragma unroll
            for (int ii = 0; ii < 4; ii++) {
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride) {
                    int gradWeightIndex = weightRow + featureDim;
                    gradWeight[gradWeightIndex] = weight[ii];
                }
            }
            
            idx++;
        } while (idx < numel && input[idx] == input[idx - 1]);
    }
}