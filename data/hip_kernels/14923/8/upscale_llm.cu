#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK_X 16
#define THREADS_PER_BLOCK_Y 16

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor)
{
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void upscale(float *input, float *output, long no_elements, int scale_factor, int d1, int d2, int d3)
{
    // Optimized block offset calculation
    long ii = threadIdx.x + blockIdx.x * THREADS_PER_BLOCK_X;
    ii += (threadIdx.y + blockIdx.y * THREADS_PER_BLOCK_Y) * (THREADS_PER_BLOCK_X * gridDim.x);
    
    // Boundary check
    if (ii >= no_elements) return;
    
    // Calculate the input index
    int ipidx = translate_idx(ii, d1, d2, d3, scale_factor);
    
    // Upscale operation
    output[ii] = input[ipidx];
}
```
