#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_MSECriterion_updateOutput_kernel(float* output, float *input, float *target, int nframe, int dim, int sizeAverage)
{
    __shared__ float buffer[MSECRITERION_THREADS];
    int k = blockIdx.x;
    float *input_k = input + k * dim;
    float *target_k = target + k * dim;

    int i_start = threadIdx.x;
    int i_step = blockDim.x;

    // Initialize buffer to 0
    buffer[threadIdx.x] = 0.0f;
    for (int i = i_start; i < dim; i += i_step)
    {
        float z = input_k[i] - target_k[i];
        buffer[threadIdx.x] += z * z;
    }
    __syncthreads();

    // Improve reduction using parallel reduction within shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            buffer[threadIdx.x] += buffer[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Write result to output
    if (threadIdx.x == 0)
    {
        *output = buffer[0];
        if (sizeAverage)
            *output /= dim;
    }
}