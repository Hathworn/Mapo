#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_MSECriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, float norm, int nframe, int dim)
{
    int k = blockIdx.x;
    float *gradInput_k = gradInput + k * dim;
    float *input_k = input + k * dim;
    float *target_k = target + k * dim;

    int i = threadIdx.x;
    int totalThreads = blockDim.x * gridDim.x;
    // Unroll the loop to increase performance and ensure memory coalescing
    for (int idx = i; idx < dim; idx += totalThreads)
    {
        gradInput_k[idx] = norm * (input_k[idx] - target_k[idx]);
    }
}