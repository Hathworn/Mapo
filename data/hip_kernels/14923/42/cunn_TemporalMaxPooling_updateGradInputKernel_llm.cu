#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate global indices for threads
    int batch_idx = blockIdx.x;
    int time_idx = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    
    if (time_idx < output_w) {
        // Compute base pointers for current batch
        float *gradInput_data = gradInput + batch_idx * input_w * input_n + time_idx * input_n * dW;
        float *gradOutput_data = gradOutput + batch_idx * output_w * input_n + time_idx * input_n;
        float *indices_data = indices + batch_idx * output_w * input_n + time_idx * input_n;

        // Unroll loop over input_n for better performance
        for (int feat = 0; feat < input_n; ++feat) {
            int idx = static_cast<int>(indices_data[feat]) * input_n + feat;
            // Accumulate gradient
            atomicAdd(&gradInput_data[idx], gradOutput_data[feat]);
        }
    }
}