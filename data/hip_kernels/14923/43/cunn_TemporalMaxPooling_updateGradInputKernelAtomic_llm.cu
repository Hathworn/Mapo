#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(
    float *gradInput, float *gradOutput, float *indices, int input_w, 
    int input_n, int output_w, int kW, int dW) 
{
    // Precompute starting indices for gradInput, gradOutput, and indices
    int timeIndex = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    if (timeIndex < output_w) {
        int batchIndex = blockIdx.x * output_w * input_n + timeIndex * input_n;
        int inputBatchIdx = blockIdx.x * input_w * input_n + timeIndex * input_n * dW;
        
        float *gradInput_data = gradInput + inputBatchIdx;
        float *gradOutput_data = gradOutput + batchIndex;
        float *indices_data = indices + batchIndex;

        // Use a single loop to process features
        for (int feat = 0; feat < input_n; ++feat) {
            float idx = indices_data[feat];
            atomicAdd(&gradInput_data[(int)idx * input_n + feat], gradOutput_data[feat]);
        }
    }
}