#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinput(float *gradInput, float *gradOutput, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx = threadIdx.x;
    int yy = threadIdx.y + blockDim.y * blockIdx.y;

    // select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;

    // compute gradInput
    if (yy < output_h) {  // Ensure within bounds
        for (; xx < output_w; xx += blockDim.x) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput;
            
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    ptr_gradInput[kx] += z / float(kW * kH);  // Update gradInput
                }
                ptr_gradInput += input_w;
            }
        }
    }
}