#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SOFTMAX_THREADS 1024

__global__ void cunn_SoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int nframe, int dim, int stride)
{
    __shared__ float buffer[SOFTMAX_THREADS];
    float *gradInput_k = gradInput + blockIdx.x * dim * stride + blockIdx.y;
    float *output_k = output + blockIdx.x * dim * stride + blockIdx.y;
    float *gradOutput_k = gradOutput + blockIdx.x * dim * stride + blockIdx.y;

    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;

    // Initialize buffer with 0
    float local_sum = 0.0f;
    for (int i = i_start; i < i_end; i += i_step)
    {
        local_sum += gradOutput_k[i * stride] * output_k[i * stride];
    }
    buffer[threadIdx.x] = local_sum; // Store local sum to shared memory

    __syncthreads();

    // Perform parallel reduction
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if (threadIdx.x < offset)
        {
            buffer[threadIdx.x] += buffer[threadIdx.x + offset];
        }
        __syncthreads();
    }

    float sum_k = buffer[0]; // The reduced sum from all threads
    for (int i = i_start; i < i_end; i += i_step)
    {
        gradInput_k[i * stride] = output_k[i * stride] * (gradOutput_k[i * stride] - sum_k);
    }
}