#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradweight(float *input, float *gradOutput, float *gradWeight, float *gradBias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW, float scale)
{
    // output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;

    int yy_start = threadIdx.y;
    int yy_step = blockDim.y;

    // select input/output plane
    gradOutput += o * output_w * output_h;
    input += i * input_w * input_h;

    // thread ID
    int tid = blockDim.x * threadIdx.y + threadIdx.x;

    // create array to hold partial sums
    __shared__ float sums[CUDA_MAX_THREADS];
    sums[tid] = 0;

    // compute partial sums
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            float *ptr_input = input + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput;
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    sums[tid] += z * ptr_input[kx];
                }
                ptr_input += input_w;
            }
        }
    }
    __syncthreads();

    // reduce: accumulate all partial sums to produce final gradWeight
    if (tid == 0) {  // single thread reduction
        for (int i = 0; i < blockDim.x * blockDim.y; i++) gradWeight[k] += scale * sums[i];
    }
    __syncthreads();

    // compute gradBias
    sums[tid] = 0;
    for (int i = tid; i < output_w * output_h; i += (blockDim.x * blockDim.y)) {
        sums[tid] += gradOutput[i];
    }
    __syncthreads();

    // reduce gradBias
    if (tid == 0) {  // single thread reduction
        for (int i = 0; i < (blockDim.x * blockDim.y); i++)
            gradBias[k] += scale * sums[i];
    }
}