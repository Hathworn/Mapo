#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv( int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ __forceinline__ size_t idx(const size_t nc, const size_t height, const size_t width, const size_t y, const size_t x) {
    return (nc * height + y) * width + x;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x;
    ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
    if (ii >= no_elements) return;
    
    // Precompute new d2 and d3 to avoid redundant calculations in the loop
    int d2_scaled = d2 * scale_factor;
    int d3_scaled = d3 * scale_factor;

    // Use shared memory to store gradOutput_data parts if required for further optimization
    __shared__ float shared_gradOutput[BLOCK_SIZE]; // Example: Define BLOCK_SIZE depending on application

    for (int i = 0; i < scale_factor; i++) {
        for(int j = 0; j < scale_factor; j++) {
            int ipidx = translate_idx_inv(ii, d1, d2_scaled, d3_scaled, scale_factor, i, j);
            // Optionally load into shared memory
            // shared_gradOutput[threadIdx.x] = gradOutput_data[ipidx];
            // __syncthreads();

            // Then use shared memory in operations
            gradInput_data[ii] += gradOutput_data[ipidx];
            // gradInput_data[ii] += shared_gradOutput[threadIdx.x];
        }
    }
}