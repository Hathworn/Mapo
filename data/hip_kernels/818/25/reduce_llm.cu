#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce(int *a, int *res) {
    __shared__ int cache[threadsPerBlock];
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    
    cache[cacheIndex] = a[tid];
    
    __syncthreads();
    
    // Use a more efficient loop with division instead of multiple if statements
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }
    
    if (cacheIndex == 0) {
        *res = cache[0];
    }
}