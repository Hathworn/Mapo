#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_matrix_mult_one(int *a, int *b, int *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y; // get the row
    int col = blockIdx.x * blockDim.x + threadIdx.x; // get the column
    int sum = 0; // initialize the sum

    // Load each row and column element into shared memory for reuse
    __shared__ int shared_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int shared_b[BLOCK_SIZE][BLOCK_SIZE];

    if (col < k && row < m) // check to make sure that the thread needs to compute
    {
        for (int tile = 0; tile < (n + BLOCK_SIZE - 1) / BLOCK_SIZE; ++tile) 
        {
            // Each thread loads one element from matrices
            if (tile * BLOCK_SIZE + threadIdx.x < n && row < m)
                shared_a[threadIdx.y][threadIdx.x] = a[row * n + tile * BLOCK_SIZE + threadIdx.x];
            else
                shared_a[threadIdx.y][threadIdx.x] = 0;

            if (tile * BLOCK_SIZE + threadIdx.y < n && col < k)
                shared_b[threadIdx.y][threadIdx.x] = b[(tile * BLOCK_SIZE + threadIdx.y) * k + col];
            else
                shared_b[threadIdx.y][threadIdx.x] = 0;

            __syncthreads(); // Synchronize to make sure submatrices are loaded

            // Multiply the two matrices
            for (int i = 0; i < BLOCK_SIZE; ++i)
                sum += shared_a[threadIdx.y][i] * shared_b[i][threadIdx.x];

            __syncthreads(); // Synchronize to make sure computation is done before loading new submatrices
        }
        c[row * k + col] = sum; // Write the block submatrix to global memory
    }
}
```
Note: The above optimization assumes the definition of `BLOCK_SIZE` somewhere else in the code, which matches the CUDA block size used. Adjust `BLOCK_SIZE` accordingly to best fit your hardware characteristics.