#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void big_add(int *a, int *b, int *c, unsigned int N) {
    // Compute global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Prefetch data assuming N is a multiple of stride
    for (; tid < N; tid += stride) {
        c[tid] = a[tid] + b[tid];
    }
}