#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void big_add(int *a, int *b, int *c, unsigned int N) {
    // Improved loading of thread id and use const stride
    const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    for (unsigned int i = tid; i < N; i += stride) {
        c[i] = a[i] + b[i];
    }
}