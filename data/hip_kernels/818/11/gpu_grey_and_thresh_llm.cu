#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_grey_and_thresh(unsigned char* Pout, unsigned char* Pin, int width, int height){
  
  int channels = 3;
  unsigned char thresh = 157;
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  int row = threadIdx.y + blockIdx.y * blockDim.y;

  // Improved: Check pixel bounds early
  if (col >= width || row >= height) return;

  int gOffset = row * width + col;
  int rgbOffset = gOffset * channels;

  // Load RGB values once
  unsigned char r = Pin[rgbOffset];
  unsigned char g = Pin[rgbOffset + 1];
  unsigned char b = Pin[rgbOffset + 2];

  // Calculate grey value using integer arithmetic
  unsigned char gval = static_cast<unsigned char>(0.21f * r + 0.71f * g + 0.07f * b);

  // Use ternary operator for simplicity
  Pout[gOffset] = gval > thresh ? 255 : 0;
}