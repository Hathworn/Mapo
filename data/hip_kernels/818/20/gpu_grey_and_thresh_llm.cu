#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_grey_and_thresh(unsigned char* Pout, unsigned char* Pin, int width, int height) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Check if pixel within range
    if (col < width && row < height) {
        int gOffset = row * width + col;
        int rgbOffset = gOffset * 3;  // Channels are constant, simplified calculation.
        
        // Load values from global memory once, reuse values
        unsigned char r = Pin[rgbOffset];
        unsigned char g = Pin[rgbOffset + 1];
        unsigned char b = Pin[rgbOffset + 2];
        
        // Apply weighted sum to convert to grayscale
        unsigned char gval = __fmaf_rn(0.21f, r, __fmaf_rn(0.71f, g, 0.07f * b)); // Use fast multiply-add operations
        
        // Apply threshold and write to output
        Pout[gOffset] = gval > 157 ? 255 : 0;  // Inline threshold check for brevity
    }
}