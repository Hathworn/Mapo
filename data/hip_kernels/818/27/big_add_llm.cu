#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void big_add(int *a, int *b, int *c, unsigned int N) {
    // Calculate unique index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop 4 times for better efficiency
    int stride = blockDim.x * gridDim.x * 4;
    for (; tid + 3 < N; tid += stride) {
        c[tid] = a[tid] + b[tid];
        c[tid + 1] = a[tid + 1] + b[tid + 1];
        c[tid + 2] = a[tid + 2] + b[tid + 2];
        c[tid + 3] = a[tid + 3] + b[tid + 3];
    }

    // Handle remaining elements
    for (; tid < N; tid += blockDim.x * gridDim.x) {
        c[tid] = a[tid] + b[tid];
    }
}