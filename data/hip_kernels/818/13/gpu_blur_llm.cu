#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_blur(unsigned char* Pout, unsigned char* Pin, int width, int height) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int k_size = 1; // Update kernel size as +1 to include only center pixels

    if (col < width && row < height) {
        int pixVal = 0;
        int pixels = 0;

        // Use loop unrolling for performance optimization
        #pragma unroll
        for (int blurRow = -k_size; blurRow <= k_size; blurRow++) {
            #pragma unroll
            for (int blurCol = -k_size; blurCol <= k_size; blurCol++) {
                int curRow = row + blurRow;
                int curCol = col + blurCol;

                if (curRow >= 0 && curRow < height && curCol >= 0 && curCol < width) {
                    pixVal += Pin[curRow * width + curCol];
                    pixels++;
                }
            }
        }

        Pout[row * width + col] = static_cast<unsigned char>(pixVal / pixels); // Ensure proper casting
    }
}