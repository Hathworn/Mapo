#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_blur(unsigned char* Pout, unsigned char* Pin, int width, int height) {
    // Calculate row and column indices
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int k_size = 3;

    // Check bounds to ensure within image dimensions
    if (col < width && row < height) {
        int pixVal = 0;
        int pixels = 0;

        // Loop over kernel size applying bounds check logic directly
        for (int blurRow = max(0, row - k_size); blurRow <= min(row + k_size, height - 1); blurRow++) {
            for (int blurCol = max(0, col - k_size); blurCol <= min(col + k_size, width - 1); blurCol++) {
                pixVal += Pin[blurRow * width + blurCol];
                pixels++;
            }
        }

        // Calculate average pixel value
        Pout[row * width + col] = (unsigned char)(pixVal / pixels);
    }
}