#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_blur(unsigned char* Pout, unsigned char* Pin, int width, int height) {
    // Calculate thread's column and row within the image
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int k_size = 3;

    if (col < width && row < height) {
        int pixVal = 0;
        int pixels = 0;
        
        // Efficiently compute the boundary of the blur kernel
        int rowStart = max(0, row - k_size);
        int rowEnd = min(height - 1, row + k_size);
        int colStart = max(0, col - k_size);
        int colEnd = min(width - 1, col + k_size);
        
        for (int curRow = rowStart; curRow <= rowEnd; curRow++) {
            for (int curCol = colStart; curCol <= colEnd; curCol++) {
                pixVal += Pin[curRow * width + curCol];
                pixels++;
            }
        }
        
        // Calculate the result pixel value
        Pout[row * width + col] = (unsigned char)(pixVal / pixels);
    }
}