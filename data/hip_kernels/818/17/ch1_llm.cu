#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ch1(unsigned char* Pout, const unsigned char* __restrict__ Pin, int width, int height) {
    int channels = 3;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure the thread is processing a valid pixel
    if (col < width && row < height) {
        int gOffset = row * width + col;
        int rgbOffset = gOffset * channels;

        // Use shared memory to leverage memory coalescing
        unsigned char r = Pin[rgbOffset];
        unsigned char g = Pin[rgbOffset + 1];
        unsigned char b = Pin[rgbOffset + 2];

        // Calculate grayscale value
        Pout[gOffset] = __float2int_rn(0.21f * r + 0.71f * g + 0.07f * b);
    }
}