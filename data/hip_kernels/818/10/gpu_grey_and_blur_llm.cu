#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_grey_and_blur(unsigned char* Pout, unsigned char* Pin, int width, int height) {
    int channels = 3;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    // Check if pixel within range for greyscale conversion
    if (col < width && row < height) {
        int gOffset = row * width + col;
        int rgbOffset = gOffset * channels;
        unsigned char r = Pin[rgbOffset];
        unsigned char g = Pin[rgbOffset + 1];
        unsigned char b = Pin[rgbOffset + 2];
        Pout[gOffset] = 0.21f * r + 0.71f * g + 0.07f * b;
    }
    __syncthreads();

    // Apply blur using shared memory
    __shared__ unsigned char sharedMem[32][32];  // Modify the size based on blockDim for best performance
    if (col < width && row < height) {
        sharedMem[threadIdx.y][threadIdx.x] = Pout[row * width + col];
    }
    __syncthreads();

    unsigned char k_size = 1;
    int pixVal = 0;
    int pixels = 0;
    for(int blurRow = -k_size; blurRow <= k_size; ++blurRow) {
        for(int blurCol = -k_size; blurCol <= k_size; ++blurCol) {
            int curRow = threadIdx.y + blurRow;
            int curCol = threadIdx.x + blurCol;

            if (curRow >= 0 && curRow < blockDim.y && curCol >= 0 && curCol < blockDim.x) {
                pixVal += sharedMem[curRow][curCol];
                pixels++;
            }
        }
    }
    __syncthreads();

    if (col < width && row < height) {
        Pout[row * width + col] = (unsigned char)(pixVal / pixels);
    }
}