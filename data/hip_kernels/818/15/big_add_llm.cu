#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void big_add(int *a, int *b, int *c, unsigned int N)
{
    // Calculate the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by using shared memory if possible (not applicable here as it's a direct add)
    // Loop over the array with increments of the total number of threads
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        c[i] = a[i] + b[i]; // Addition operation
    }
}