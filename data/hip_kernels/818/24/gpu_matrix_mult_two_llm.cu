#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_matrix_mult_two(int *d_M, int *d_N, int *d_P, int m, int n, int k)
{
    // shared memory for tiling
    __shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ int Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;  
    int by = blockIdx.y;
    int tx = threadIdx.x; 
    int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    int pval = 0;

    for (int ph = 0; ph < n / TILE_WIDTH; ph++) {
        // Optimized boundary checking
        if (row < k) {
            Mds[ty][tx] = (ph * TILE_WIDTH + tx < m) ? d_M[row * k + ph * TILE_WIDTH + tx] : 0;
        }
        if (col < m) {
            Nds[ty][tx] = (ph * TILE_WIDTH + ty < k) ? d_N[(ph * TILE_WIDTH + ty) * k + col] : 0;
        }

        __syncthreads();  // execution barrier

        // Unrolling loop for performance optimization
        #pragma unroll
        for (int j = 0; j < TILE_WIDTH; j++) {
            pval += Mds[ty][j] * Nds[j][tx];
        }

        __syncthreads();
    }

    if (row < k && col < m) {
        d_P[row * k + col] = pval;
    }
}