#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_matrix_mult(int *a, int *b, int *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Get the row index
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Get the column index
    int sum = 0; // Initialize the sum

    if (col < k && row < m) // Check boundary conditions
    {
        // Use shared memory for better data reuse
        __shared__ int shared_a[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ int shared_b[BLOCK_SIZE][BLOCK_SIZE];
        
        for (int tileIdx = 0; tileIdx < (n + BLOCK_SIZE - 1) / BLOCK_SIZE; ++tileIdx)
        {
            if (tileIdx * BLOCK_SIZE + threadIdx.x < n && row < m)
                shared_a[threadIdx.y][threadIdx.x] = a[row * n + tileIdx * BLOCK_SIZE + threadIdx.x];
            else
                shared_a[threadIdx.y][threadIdx.x] = 0;

            if (tileIdx * BLOCK_SIZE + threadIdx.y < n && col < k)
                shared_b[threadIdx.y][threadIdx.x] = b[(tileIdx * BLOCK_SIZE + threadIdx.y) * k + col];
            else
                shared_b[threadIdx.y][threadIdx.x] = 0;

            __syncthreads();

            for (int i = 0; i < BLOCK_SIZE; ++i)
                sum += shared_a[threadIdx.y][i] * shared_b[i][threadIdx.x];

            __syncthreads();
        }
        c[row * k + col] = sum; // Write the result to matrix c
    }
}