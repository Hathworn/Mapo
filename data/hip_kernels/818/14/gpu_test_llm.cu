#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_test(unsigned char* Pout, unsigned char* Pin, int width, int height) {
    // Calculate global thread index in x and y direction
    int col = blockIdx.x * blockDim.x + threadIdx.x; 
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check within image boundaries before writing
    if (row < height && col < width) {
        int i = row * width + col;  // Calculate index within linear memory
        Pout[i] = Pin[i];  // Copy data from Pin to Pout
    }
}