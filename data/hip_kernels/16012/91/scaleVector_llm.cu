#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (pos < len) 
    {
        // Efficient parallel scaling of vector elements
        d_res[pos] = d_src[pos] * scale;
    }
}