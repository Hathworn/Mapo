#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    // pixel containing bottom left corner
    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);
    // target pixel integer coords
    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];

    // Pre-calculate weights
    float weight_br = dx * dy;               // bottom right
    float weight_bl = (1.0f - dx) * dy;      // bottom left
    float weight_ul = (1.0f - dx) * (1.0f - dy); // upper left
    float weight_ur = dx * (1.0f - dy);      // upper right

    // Fill pixel containing bottom right corner
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0))
    {
        _atomicAdd(dst + ty * image_stride + tx, value * weight_br);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight_br);
    }

    // Fill pixel containing bottom left corner
    tx -= 1;
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0))
    {
        _atomicAdd(dst + ty * image_stride + tx, value * weight_bl);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight_bl);
    }

    // Fill pixel containing upper left corner
    ty -= 1;
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0))
    {
        _atomicAdd(dst + ty * image_stride + tx, value * weight_ul);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight_ul);
    }

    // Fill pixel containing upper right corner
    tx += 1;
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0))
    {
        _atomicAdd(dst + ty * image_stride + tx, value * weight_ur);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight_ur);
    }
}