#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Inline function to retrieve imaginary part of a float2
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

// Optimized kernel function
__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i < h && j < w) {
        // Calculate position in image array
        const int pos = i * w + j;
        // Assign value to the image
        image[pos] = value;
    }
}