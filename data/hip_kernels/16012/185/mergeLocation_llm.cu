#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds before accessing memory
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];

        // Use shared memory for scale multiplication if multiple threads need the same value
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}