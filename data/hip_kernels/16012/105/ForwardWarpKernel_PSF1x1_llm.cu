#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure indices are within image boundaries
    if (i >= h || j >= w) return;

    int flow_idx = i * flow_stride + j;
    int image_idx = i * image_stride + j;

    // Pre-compute values
    float u_ = u[flow_idx];
    float v_ = v[flow_idx];

    float cx = u_ * time_scale + j + 1.0f;
    float cy = v_ * time_scale + i + 1.0f;

    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Validate target position before adding
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        float value = src[image_idx];
        atomicAdd(&dst[ty * image_stride + tx], value);
    }
}