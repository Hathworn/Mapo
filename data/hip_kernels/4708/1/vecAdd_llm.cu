#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(double *a, double *b, double *c, int n) {
    // Calculate total threads and thread id
    int stride = gridDim.x * blockDim.x;
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Process multiple elements per thread
    for (int i = id; i < n; i += stride)
        c[i] = a[i] + b[i];
}