#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load the accuracy value from global memory into a register
    float accValue = *accuracy;
    
    // Perform the division in register for faster operation
    accValue /= N;
    
    // Store the result back to global memory
    *accuracy = accValue;
}