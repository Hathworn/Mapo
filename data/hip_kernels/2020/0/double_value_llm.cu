#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" 
#include "includes.h"

// Kernel to double values using shared memory for improved memory access pattern
__global__ void double_value(double *x, double *y) 
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Compute global index
    // Use shared memory to reduce global memory access
    __shared__ double shared_x[256]; // Assuming block size is 256, adjust if needed
    shared_x[threadIdx.x] = x[idx]; // Load data to shared memory
    __syncthreads(); // Ensure all threads have written to shared memory
    y[idx] = 2.0 * shared_x[threadIdx.x]; // Compute result using shared memory
}