#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel with block and thread indexing
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if (idx == 0) {  // Let only the first thread perform the division
        *accuracy /= N;
    }
}