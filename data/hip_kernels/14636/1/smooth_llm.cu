#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth(unsigned char *entrada, unsigned char *saida, int n_linhas, int n_colunas) {
    int posicao = blockIdx.x * blockDim.x + threadIdx.x;
    if (posicao < n_linhas * n_colunas) {
        int sum = 0;
        int row, col;
        
        // Efficiently sum the 5x5 region using loops
        for (int i = 0; i < 5; ++i) {
            for (int j = 0; j < 5; ++j) {
                row = posicao / n_colunas + i;  // Calculate current row
                col = posicao % n_colunas + j;  // Calculate current column

                // Add boundary check to avoid out-of-bounds accesses
                if (row < n_linhas && col < n_colunas) {
                    sum += entrada[row * n_colunas + col];
                }
            }
        }

        // Calculate the average
        saida[posicao] = sum / 25;
    }
}