#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Use shared memory for cooperative loading of normalization_factor
    __shared__ float sharedScale[1024];  // Adjust size based on block size
    int localId = threadIdx.y * blockDim.x + threadIdx.x;

    if (i < h && j < w)
    {
        const int pos = i * s + j;

        // Load scale factor into shared memory
        sharedScale[localId] = normalization_factor[pos];
        __syncthreads();  // Synchronize to ensure complete loading

        // Use shared memory to avoid multiple global memory reads
        float scale = sharedScale[localId];
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

        image[pos] *= invScale;
    }
}