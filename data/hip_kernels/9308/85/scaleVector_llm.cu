#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Calculate the global index
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we operate within bounds
    if (pos < len)
    {
        // Perform vector scaling
        d_res[pos] = d_src[pos] * scale;
    }
}