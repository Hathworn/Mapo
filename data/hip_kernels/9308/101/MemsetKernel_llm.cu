#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Ensure index bounds checking is done before calculations 
    if (i < h && j < w)
    {
        const int pos = i * w + j;

        // Use a direct assignment to the memory
        image[pos] = value;
    }
}