#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Precompute common expressions
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);

    // Precalculate coordinates and weights
    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];
    float weights[4] = {dx * dy, (1.0f - dx) * dy, (1.0f - dx) * (1.0f - dy), dx * (1.0f - dy)};
    int coords[4][2] = {{tx, ty}, {tx - 1, ty}, {tx - 1, ty - 1}, {tx, ty - 1}};
    
    // Loop unrolling for improved performance
    #pragma unroll
    for (int k = 0; k < 4; ++k) {
        int x = coords[k][0];
        int y = coords[k][1];
        if (!(x >= w || x < 0 || y >= h || y < 0)) {
            _atomicAdd(dst + y * image_stride + x, value * weights[k]);
            _atomicAdd(normalization_factor + y * image_stride + x, weights[k]);
        }
    }
}