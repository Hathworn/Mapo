#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Maintain the imag function as is

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

// Optimized kernel function
__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Calculate indices
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;

    // Check boundaries
    if (i >= h || j >= w) return;

    // Calculate row offsets once
    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Load flows
    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Calculate the warped pixel position
    float cx = u_ * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v_ * time_scale + static_cast<float>(i) + 1.0f;

    // Determine target pixel
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Retrieve source value
    float value = src[image_row_offset + j];

    // Use atomic add to accumulate the value conditionally
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        // Ensure atomic addition only within valid bounds
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}