#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void OPT_2(int *d_adjList, int *d_sizeAdj, int *d_lcmMatrix, int *d_LCMSize, int n_vertices)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    // Launch threads only necessary for processing
    if (i < n_vertices)
    {
        int indexUsed = 0, indexOffset = 0;
        int iStart = 0, iEnd = 0;
        int k = 0;

        if (i > 0)
        {
            k = d_sizeAdj[i - 1];
            indexOffset = d_LCMSize[i - 1];
        }

        iEnd = d_sizeAdj[i];

        // Use memset to initialize matrix if supported
        for (int j = indexOffset; j < iEnd; j++)
        {
            d_lcmMatrix[j] = 0;
        }

        __syncthreads(); // Ensure memory consistency

        for (int j = 0; j < n_vertices; j++) {
            if (i == j)
                continue;
            iStart = k;
            int jStart = 0, jEnd = 0;

            if (j > 0)
                jStart = d_sizeAdj[j - 1];
            jEnd = d_sizeAdj[j];

            int compVec = 0;

            while (iStart < iEnd && jStart < jEnd)
            {
                if (d_adjList[iStart] < d_adjList[jStart])
                    iStart++;
                else if (d_adjList[jStart] < d_adjList[iStart])
                    jStart++;
                else // if arr1[i] == arr2[j]
                {
                    jStart++;
                    iStart++;
                    compVec++;
                }
            }

            if (compVec > 0)
            {
                atomicAdd((int *)&d_lcmMatrix[indexUsed + indexOffset], compVec);
                indexUsed++;
            }
        }

        // Use atomic operations for LCMSize update to avoid synchronization
        atomicExch(&d_LCMSize[i], indexUsed);
    }
}