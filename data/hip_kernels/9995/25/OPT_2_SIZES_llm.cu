#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void OPT_2_SIZES(int *d_adjList, int *d_sizeAdj, int *d_LCMSize, int n_vertices)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < n_vertices)
    {
        int indexUsed = 0;
        int iStart = i == 0 ? 0 : d_sizeAdj[i-1];
        int iEnd = d_sizeAdj[i];

        // __syncthreads(); // Removed redundant __syncthreads() as data isn't shared across threads

        for(int j = 0; j < n_vertices; j++) 
        {
            if(i == j) continue;
            int jStart = j == 0 ? 0 : d_sizeAdj[j-1];
            int jEnd = d_sizeAdj[j];

            int compVec = 0;

            int iPtr = iStart, jPtr = jStart;
            // Loop through both adjacency lists
            while (iPtr < iEnd && jPtr < jEnd)
            {
                if(d_adjList[iPtr] < d_adjList[jPtr])
                    iPtr++;
                else if (d_adjList[jPtr] < d_adjList[iPtr])
                    jPtr++;
                else
                {
                    jPtr++;
                    iPtr++;
                    compVec++;
                    // Early exit if needed
                }
            }

            if (compVec > 0)
            {
                indexUsed++;
            }
        }

        // __syncthreads(); // Removed as it's not needed here

        d_LCMSize[i] = indexUsed;
    }
}
```
