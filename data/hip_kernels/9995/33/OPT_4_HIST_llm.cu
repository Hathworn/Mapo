#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void OPT_4_HIST(int *d_lcmMatrix, int *d_LCMSize, int *d_histogram, int n_vertices)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n_vertices)
    {
        int iStart = (i > 0) ? d_LCMSize[i - 1] : 0; // Use ternary operator for efficiency
        int iSize = d_LCMSize[i] - iStart;
        int count = 0;
        
        // Loop unrolling can be manually applied here if iSize is known to be small and bounded
        for (int j = 0; j < n_vertices; j++) 
        {
            int jStart = (j > 0) ? d_LCMSize[j - 1] : 0; // Use ternary operator for efficiency
            int jSize = d_LCMSize[j] - jStart;
            if (iSize != jSize)
                continue;

            // Optimize: Replace loop with memory comparison if possible
            bool eq = true;
            for (int k = 0; k < iSize; k++)
            {
                if (d_lcmMatrix[iStart + k] != d_lcmMatrix[jStart + k])
                {
                    eq = false;
                    break;
                }
            }
            
            if (eq) 
            {
                count++;
            }
        }

        // Use atomicAdd for thread-safe histogram updates
        atomicAdd(&d_histogram[count], 1);
    }
}