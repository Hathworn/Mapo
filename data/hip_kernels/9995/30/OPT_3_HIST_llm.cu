#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void OPT_3_HIST(int* lcm, int* hist, int* lcmsizes, int n) {
    int vertex = blockIdx.x;
    int vcomp = threadIdx.x;

    // Use shared memory for cval
    __shared__ int cval;

    if (vcomp == 0) cval = 0;
    __syncthreads();

    if (vertex < n) {
        for (int i = vcomp; i < n; i += blockDim.x) {
            bool equal = false;

            if (vertex == i) {
                atomicAdd(&cval, 1);
            }
            else if ((lcmsizes[vertex+1] - lcmsizes[vertex]) == (lcmsizes[i+1] - lcmsizes[i])) {
                equal = true;

                // Use loop unrolling for comparison
                for (int j = 0; j < lcmsizes[vertex+1] - lcmsizes[vertex]; j++) {
                    if (lcm[lcmsizes[vertex] + j] != lcm[lcmsizes[i] + j]) {
                        equal = false;
                        break;
                    }
                }

                if (equal) {
                    atomicAdd(&cval, 1);
                }
            }
        }
    }
    __syncthreads();

    // Use atomic operations carefully and minimize them
    if (vcomp == 0 && cval > 0) {
        atomicAdd(&hist[cval], 1);
        // printf("\nv%d: %d\n", vertex, cval); // Uncomment for debugging
    }
}