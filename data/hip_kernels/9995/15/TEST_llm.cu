#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TEST(int n, float* x, float* y) {
    // Use thread index to parallelize calculations, improving efficiency
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        y[i] += x[i];
}