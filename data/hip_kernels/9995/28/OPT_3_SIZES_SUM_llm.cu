#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void OPT_3_SIZES_SUM(int* lcmsizes, int n) {
    // Use thread index for parallel addition
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) atomicAdd(&lcmsizes[i+1], lcmsizes[i]);
}