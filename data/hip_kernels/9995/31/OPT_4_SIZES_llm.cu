#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void OPT_4_SIZES(int *d_adjList, int *d_sizeAdj, int *d_LCMSize, int n_vertices)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < n_vertices)
    {
        int indexUsed = 0;
        int iStart = 0;
        int iEnd = d_sizeAdj[i];

        if(i > 0) 
        {
            iStart = d_sizeAdj[i-1];
        }

        // Remove unnecessary __syncthreads()

        for(int j = 0; j < n_vertices; j++) 
        {
            if(i == j) 
                continue;

            int jStart = 0;
            int jEnd = d_sizeAdj[j];

            if(j > 0)
                jStart = d_sizeAdj[j-1];

            int compVec = 0; 

            // Optimize the while loop for better memory access patterns
            for (; iStart < iEnd && jStart < jEnd; )
            {
                int val1 = d_adjList[iStart];
                int val2 = d_adjList[jStart];
                
                if(val1 < val2)
                    iStart++;
                else if (val2 < val1)
                    jStart++;
                else
                {
                    compVec = 1;
                    break;
                }
            }

            if (compVec > 0)
            {
                indexUsed++;
            }
        }

        // Remove unnecessary __syncthreads()
        
        d_LCMSize[i] = indexUsed;
    }
}