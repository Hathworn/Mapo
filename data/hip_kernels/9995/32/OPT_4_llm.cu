#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void OPT_4(int *d_adjList, int *d_sizeAdj, int *d_lcmMatrix, int *d_LCMSize, int n_vertices)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < n_vertices)
    {
        int indexUsed = 0;
        int indexOffset = 0;
        int iStart = 0;
        int iEnd = 0;
        int k = 0;

        if(i > 0)
        {
            k = d_sizeAdj[i-1];
            indexOffset = d_LCMSize[i-1];
        }
        
        iEnd = d_sizeAdj[i];

        for(int j = indexOffset; j < iEnd; j++)
        {
            d_lcmMatrix[j] = 0;
        }

        __syncthreads(); // Ensure matrices are zeroed before comparison

        for(int j = 0; j < n_vertices; j++) {
            if(i == j) continue;
            iStart = k;
            int jStart = 0, jEnd = 0;

            if(j > 0)
                jStart = d_sizeAdj[j-1];
            jEnd = d_sizeAdj[j];

            int compVec = 0;

            // Optimize using while loop with early breakout condition
            while (iStart < iEnd && jStart < jEnd)
            {
                if(d_adjList[iStart] < d_adjList[jStart])
                    iStart++;
                else if (d_adjList[jStart] < d_adjList[iStart])
                    jStart++;
                else
                {
                    jStart++;
                    iStart++;
                    compVec++;
                }
            }

            if (compVec > 0)
            {
                atomicAdd(&d_lcmMatrix[indexUsed + indexOffset], compVec); // Atomic operation for memory safety
                indexUsed++;
            }
        }

        // Store the number of common elements for this vertex
        d_LCMSize[i] = indexUsed;
    }
}