#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Naive_Hist(int* d_result, int* d_hist, int n_vertices) {
    int row = blockIdx.x;
    int row2 = threadIdx.x;

    __shared__ int count; // Shared counter for each block

    // Initialize count to zero using the first thread of the block
    if (row2 == 0) count = 0;
    __syncthreads();

    // Loop through vertices with a stride of blockDim.x for better GPU utilization
    if (row < n_vertices) {
        for (int i = row2; i < n_vertices; i += blockDim.x) {
            bool equal = true;
            for (int j = 0; j < n_vertices; j++) {
                if (d_result[row * n_vertices + j] != d_result[i * n_vertices + j]) {
                    equal = false;
                    break;
                }
            }
            // Use atomic function to safely update shared count
            if (equal) atomicAdd(&count, 1);
        }
    }

    // Synchronization to ensure all threads have updated the count
    __syncthreads();

    // Update histogram atomically
    if (row < n_vertices && row2 == 0 && count > 0) {
        atomicAdd(&d_hist[count], 1);
    }
}