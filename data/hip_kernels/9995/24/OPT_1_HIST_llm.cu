#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void OPT_1_HIST(int* lcm, int* hist, int n) {

    int vertex = blockIdx.x;
    int vcomp = threadIdx.x;

    // Shared memory for storing result of each block
    __shared__ int cval[1];

    // Initialize shared memory
    if(vcomp == 0) {
        cval[0] = 0;
    }
    __syncthreads();

    // Iterate over possible comparisons
    if(vertex < n && vcomp < n) {
        for(int i = vcomp; i < n; i += blockDim.x) {

            bool equal = false;

            if(vertex == i) {
                atomicAdd(&cval[0], 1);
            } else {
                equal = true;

                #pragma unroll // Use loop unrolling for better performance
                for(int j = 0; j < n; j++) {
                    if(lcm[vertex * n + j] != lcm[i * n + j]) {
                        equal = false;
                        break;
                    }
                }

                if(equal) {
                    atomicAdd(&cval[0], 1);
                }
            }
        }
    }

    __syncthreads();
    if(vertex < n && vcomp == 0 && cval[0] > 0) {
        atomicAdd(&hist[cval[0]], 1);
    }
}