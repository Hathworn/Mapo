#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void OPT_3_SIZES(int* adj, int* lcmsizes, int* sizes, int n) {
    // Cache vertex size to minimize repetitive global memory access
    extern __shared__ int vertexSizes[];
    
    int vertex = blockIdx.x;
    int vcomp = threadIdx.x;

    if (vertex < n && vcomp < n) {
        int vertexStart = sizes[vertex];
        int vertexEnd = sizes[vertex + 1];
        int vertexSize = vertexEnd - vertexStart;

        for (int i = vcomp; i < n; i += blockDim.x) {
            if (vertex == i) continue;

            // Cache comparison vertex size to minimize repetitive global memory access
            int vcompStart = sizes[i];
            int vcompEnd = sizes[i + 1];
            int vcompSize = vcompEnd - vcompStart;
            
            int cval = 0;

            // Check adjacency with shared memory and loop unrolling for efficiency
#pragma unroll 4
            for (int j = 0; j < vertexSize; j++) {
#pragma unroll 4
                for (int k = 0; k < vcompSize; k++) {
                    if (adj[vertexStart + j] == adj[vcompStart + k]) {
                        cval = 1;  // Simplify and use flag check
                        break;
                    }
                }
                if (cval) {
                    atomicAdd(&lcmsizes[vertex + 1], 1);
                    break;
                }
            }
        }
    }
}