#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TEST(int n, float* x, float* y) {
    // Use block and thread indices to allow parallel processing
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure only valid indices are used
    if (i < n) {
        y[i] += x[i];
    }
}