#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yMinDeltaIntegralKernel(const float *intData, const int intDataStrideChannel, float *tmpArray, const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w, const float *xMin, const float *xMax, const float *yMin) {

    int id = NUM_THREADS * blockIdx.x + threadIdx.x;
    tmpArray += id; // Adjust output location
  
    const int y = id % w + 1; id /= w; // Calculate y-coordinate (1-indexed)
    const int x = id % h + 1; id /= h; // Calculate x-coordinate (1-indexed)
    const int windowIdx = id % nWindows; id /= nWindows; // Determine window index
  
    intData += id * intDataStrideChannel; // Move pointer to relevant input data
  
    const int globalWindowIdx = (id % nInputPlane) * nWindows + windowIdx; id /= nInputPlane;
    const int & batchIdx = id;
  
    if (batchIdx < batchSize) {
        const int xMinInt = (int)ceil(xMin[globalWindowIdx] - 1);
        const int yMinInt = (int)ceil(yMin[globalWindowIdx] - 1);
        const int xMaxInt = (int)floor(xMax[globalWindowIdx]);
    
        float delta = 0;
    
        // Compute integral values with bounds checking
        int xMaxBound = max(0, min(x + xMaxInt, h));
        int xMinBound = max(0, min(x + xMinInt, h));
        int yMinBound = max(0, min(y + yMinInt, w));
        if (y + yMinInt >= 1 && y + yMinInt < w) { // Check for valid y-coordinate
            delta += intData[xMaxBound * (w+1) + yMinBound];
            delta -= intData[xMaxBound * (w+1) + max(0, yMinBound - 1)];
            delta -= intData[xMinBound * (w+1) + yMinBound];
            delta += intData[xMinBound * (w+1) + max(0, yMinBound - 1)];
        }
        *tmpArray = -delta; // Store the calculated delta
    }
}