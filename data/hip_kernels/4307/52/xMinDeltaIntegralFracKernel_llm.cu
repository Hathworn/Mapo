#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xMinDeltaIntegralFracKernel( const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMin, const float *yMin, const float *yMax, const float *inData, const int inDataStrideRow) {

    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Use blockDim directly for better readability
    const int totalPixels = h * w;
    const int pixelIdx = id % totalPixels;  // Determine the pixel index
    id /= totalPixels;
    const int &windowIdx = id;

    if (windowIdx < nWindows) {  // Simplified boundary check

        const int x = pixelIdx / w + 1;  // Calculate x and y
        const int y = pixelIdx % w + 1;

        if (x <= h && y <= w) {

            tmpArray += windowIdx * totalPixels;
            float delta = 0.0f;

            const int rem = windowIdx % 4;
            const int idxBase = 3 * (windowIdx / 4);
            float xMinStretched = (rem == 0) ? -h : xMin[idxBase + rem - 1];
            float yMinStretched = (rem == 2) ? -w : yMin[idxBase + rem - 2];
            float yMaxStretched = (rem == 3) ? w : yMax[idxBase + rem - 3];

            const int xMinInt = (int)ceil(xMinStretched - 1);
            const int yMinInt = (int)ceil(yMinStretched - 1);
            const float yMinFrac = yMinInt - yMinStretched + 1;
            const int yMaxInt = (int)floor(yMaxStretched);
            const float yMaxFrac = yMaxStretched - yMaxInt;

            // Optimize access and calculations by avoiding redundant conditions
            if (rem != 0) {  // Calculations only needed when rem is not 0
                const float tlCorner = (y + yMinInt <  1 || x + xMinInt <  1) ? 0 :
                    inData[max(0, min(h - 1, x + xMinInt - 1)) * inDataStrideRow + max(0, min(w - 1, y + yMinInt - 1))];
                const float trCorner = (y + yMaxInt >= w || x + xMinInt <  1) ? 0 :
                    inData[max(0, min(h - 1, x + xMinInt - 1)) * inDataStrideRow + max(0, min(w - 1, y + yMaxInt))];

                delta += trCorner * ((y + yMaxInt <  1) ? 1.0f : yMaxFrac);
                delta += tlCorner * ((y + yMinInt >= w) ? 1.0f : yMinFrac);
                
                // Reduce redundant intData access by organizing logic
                int xOffset = max(0, min(x + xMinInt - 1, h));
                int yOffsetMax = max(0, min(y + yMaxInt, w));
                int yOffsetMin = max(0, min(y + yMinInt, w));

                delta += intData[max(0, min(x + xMinInt, h)) * (w + 1) + yOffsetMax];
                delta -= intData[xOffset * (w + 1) + yOffsetMax];
                delta -= intData[max(0, min(x + xMinInt, h)) * (w + 1) + yOffsetMin];
                delta += intData[xOffset * (w + 1) + yOffsetMin];

                delta *= (x + xMinInt >= 1 && x + xMinInt < h);
            }
            tmpArray[(x - 1) * w + (y - 1)] *= -delta;
        }
    }
}