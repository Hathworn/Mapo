#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void xMinDeltaIntegralReplicateKernel(const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMin, const float *yMin, const float *yMax, const int strideH, const int strideW) {
    // Calculate hOut and wOut
    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;

    // Calculate unique thread index
    const int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate yOut, xOut, and windowIdx
    const int yOut = id % wOut;
    const int xOut = (id / wOut) % hOut;
    const int windowIdx = id / (wOut * hOut);

    if (windowIdx < nWindows && xOut < hOut && yOut < wOut) {
        const int x = xOut * strideH + 1;
        const int y = yOut * strideW + 1;

        // Move tmpArray pointer for the current window
        tmpArray += windowIdx * hOut * wOut;

        // Calculate integer boundaries
        const int xMinInt = (int)ceil(xMin[windowIdx] - 1);
        const int yMinInt = (int)ceil(yMin[windowIdx] - 1);
        const int yMaxInt = (int)floor(yMax[windowIdx]);

        // Initialize delta
        float delta = 0.0f;

        // Accumulate delta based on integral image
        delta += intData[max(0, min(x + xMinInt, h - 1)) * (w + 1) + max(0, min(y + yMaxInt, w))];
        delta -= intData[max(0, min(x + xMinInt - 1, h)) * (w + 1) + max(0, min(y + yMaxInt, w))];
        delta -= intData[max(0, min(x + xMinInt, h - 1)) * (w + 1) + max(0, min(y + yMinInt, w))];
        delta += intData[max(0, min(x + xMinInt - 1, h)) * (w + 1) + max(0, min(y + yMinInt, w))];

        // Conditional multiplication with delta
        if (x + xMinInt >= 1 && x + xMinInt < h) {
            tmpArray[xOut * wOut + yOut] *= -delta;
        }
    }
}