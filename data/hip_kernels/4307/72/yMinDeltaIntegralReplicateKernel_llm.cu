#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yMinDeltaIntegralReplicateKernel(const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMin, const float *xMax, const float *yMin, const int strideH, const int strideW) {

    // Calculate hOut and wOut using efficient integer division
    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;

    // Use advanced indexing with blockIdx and threadIdx
    int id = blockIdx.x * blockDim.x + threadIdx.x; 
    const int yOut = id % wOut; id /= wOut; 
    const int xOut = id % hOut; id /= hOut; 
    const int windowIdx = id;

    if (windowIdx < nWindows && xOut < hOut && yOut < wOut) {

        const int x = xOut * strideH + 1;
        const int y = yOut * strideW + 1;

        tmpArray += windowIdx * hOut * wOut;

        const int xMinInt = (int)ceil(xMin[windowIdx] - 1);
        const int yMinInt = (int)ceil(yMin[windowIdx] - 1);
        const int xMaxInt = (int)floor(xMax[windowIdx]);

        // Eliminate redundant branching with ternary operator and max/min clamping
        float delta = 0;

        delta += intData[max(0, min(x + xMaxInt, h)) * (w + 1) + max(0, min(y + yMinInt, w))];
        delta -= intData[max(0, min(x + xMaxInt, h)) * (w + 1) + max(0, min(y + yMinInt - 1, w - 1))];
        delta -= intData[max(0, min(x + xMinInt, h)) * (w + 1) + max(0, min(y + yMinInt, w))];
        delta += intData[max(0, min(x + xMinInt, h)) * (w + 1) + max(0, min(y + yMinInt - 1, w - 1))];

        // Use boolean multiplication for conditional operations
        delta *= (y + yMinInt >= 1 && y + yMinInt < w);
        tmpArray[xOut * wOut + yOut] *= -delta;
    }
}