#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xMinDeltaIntegralReplicateFracKernel(const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMin, const float *yMin, const float *yMax, const float *inData, const int inDataStrideRow, const int strideH, const int strideW) {
    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;

    // Compute a unique thread index in the grid
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    const int yOut = id % wOut; id /= wOut;
    const int xOut = id % hOut; id /= hOut;
    const int &windowIdx = id;

    if (windowIdx < nWindows && xOut < hOut && yOut < wOut) {
        const int x = xOut * strideH + 1;
        const int y = yOut * strideW + 1;

        tmpArray += windowIdx * hOut * wOut;

        const int xMinInt = (int)ceil(xMin[windowIdx] - 1);

        const int yMinInt = (int)ceil(yMin[windowIdx] - 1);
        const float yMinFrac = yMinInt - yMin[windowIdx] + 1;

        const int yMaxInt = (int)floor(yMax[windowIdx]);
        const float yMaxFrac = yMax[windowIdx] - yMaxInt;

        // Retrieve integral values
        const float tlCorner = (y + yMinInt < 1 || x + xMinInt < 1) ? 0 :
            inData[max(0, min(h - 1, x + xMinInt - 1)) * inDataStrideRow +
                   max(0, min(w - 1, y + yMinInt - 1))];
        const float trCorner = (y + yMaxInt >= w || x + xMinInt < 1) ? 0 :
            inData[max(0, min(h - 1, x + xMinInt - 1)) * inDataStrideRow +
                   max(0, min(w - 1, y + yMaxInt))];

        float delta = 0;
        delta += trCorner * ((y + yMaxInt < 1) ? 1.0f : yMaxFrac);
        delta += tlCorner * ((y + yMinInt >= w) ? 1.0f : yMinFrac);

        delta += intData[max(0, min(x + xMinInt, h)) * (w + 1)
                         + max(0, min(y + yMaxInt, w))];
        delta -= intData[max(0, min(x + xMinInt - 1, h)) * (w + 1)
                         + max(0, min(y + yMaxInt, w))];
        delta -= intData[max(0, min(x + xMinInt, h)) * (w + 1)
                         + max(0, min(y + yMinInt, w))];
        delta += intData[max(0, min(x + xMinInt - 1, h)) * (w + 1)
                         + max(0, min(y + yMinInt, w))];

        delta *= (x + xMinInt >= 1 && x + xMinInt < h);
        tmpArray[xOut * wOut + yOut] *= -delta;
    }
}