#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yMaxDeltaIntegralKernel( const float *intData, const int intDataStrideChannel, float *tmpArray, const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w, const float *xMin, const float *xMax, const float *yMax) {

    int id = NUM_THREADS * blockIdx.x + threadIdx.x;
    tmpArray += id; // tmpArray now points to our output pixel

    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int windowIdx = id % nWindows; id /= nWindows;

    // `id` is now is now the current global input plane number
    intData  += id * intDataStrideChannel;

    const int globalWindowIdx = (id % nInputPlane) * nWindows + windowIdx; id /= nInputPlane;
    const int & batchIdx = id;

    if (batchIdx < batchSize) {

        const int xMinInt = (int)ceil(xMin[globalWindowIdx]-1);
        const int xMaxInt = (int)floor(xMax[globalWindowIdx]);
        const int yMaxInt = (int)floor(yMax[globalWindowIdx]);

        // Precalculate indices for efficiency
        int idx1 = max(0, min(x + xMaxInt, h)) * (w + 1);
        int idx2 = max(0, min(x + xMinInt, h)) * (w + 1);
        int ym1 = max(0, min(y + yMaxInt, w));
        int ym2 = max(0, min(y + yMaxInt + 1, w));

        float delta = 0;
        
        // Simplified access to intData to improve performance
        delta += intData[idx1 + ym2];
        delta -= intData[idx1 + ym1];
        delta -= intData[idx2 + ym2];
        delta += intData[idx2 + ym1];

        delta *= (y + yMaxInt >= 1 && y + yMaxInt < w);
        *tmpArray = delta;
    }
}