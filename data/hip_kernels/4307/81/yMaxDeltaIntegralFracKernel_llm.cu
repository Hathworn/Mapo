#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yMaxDeltaIntegralFracKernel(const float *intData, const int intDataStrideChannel, float *tmpArray, const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w, const float *xMin, const float *xMax, const float *yMax, const float *inData, const int inDataStrideRow, const int inDataStrideChannel) {

    int id = NUM_THREADS * blockIdx.x + threadIdx.x; 
    tmpArray += id; // tmpArray now points to our output pixel

    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int windowIdx = id % nWindows; id /= nWindows;

    intData  += id * intDataStrideChannel;
    inData   += id * inDataStrideChannel;

    const int globalWindowIdx = (id % nInputPlane) * nWindows + windowIdx; id /= nInputPlane;
    const int & batchIdx = id;

    // Guard clause for batch index
    if (batchIdx >= batchSize) {
        return;
    }

    const int xMinInt = (int)ceil(xMin[globalWindowIdx] - 1);
    const float xMinFrac = xMinInt - xMin[globalWindowIdx] + 1;

    const int xMaxInt = (int)floor(xMax[globalWindowIdx]);
    const float xMaxFrac = xMax[globalWindowIdx] - xMaxInt;

    const int yMaxInt = (int)floor(yMax[globalWindowIdx]);

    int valid;
    float delta = 0.0f;

    // Precompute conditions to avoid redundant calculations
    const bool yMaxInBounds = (y + yMaxInt < w) && (y + yMaxInt >= 0);

    if (yMaxInBounds && (x + xMinInt >= 1) && (x + xMinInt <= h)) {
        const int xMinIndex = max(0, min(h - 1, x + xMinInt - 1));
        const int yMaxIndex = max(0, min(w - 1, y + yMaxInt));
        delta += inData[xMinIndex * inDataStrideRow + yMaxIndex] * xMinFrac;
    }

    if (yMaxInBounds && (x + xMaxInt >= 0) && (x + xMaxInt < h)) {
        const int xMaxIndex = max(0, min(h - 1, x + xMaxInt));
        const int yMaxIndex = max(0, min(w - 1, y + yMaxInt));
        delta += inData[xMaxIndex * inDataStrideRow + yMaxIndex] * xMaxFrac;
    }

    delta += intData[max(0, min(x + xMaxInt, h)) * (w + 1)
                + max(0, min(y + yMaxInt + 1, w))];
    delta -= intData[max(0, min(x + xMaxInt, h)) * (w + 1)
                + max(0, min(y + yMaxInt, w))];
    delta -= intData[max(0, min(x + xMinInt, h)) * (w + 1)
                + max(0, min(y + yMaxInt + 1, w))];
    delta += intData[max(0, min(x + xMinInt, h)) * (w + 1)
                + max(0, min(y + yMaxInt, w))];

    *tmpArray = delta * yMaxInBounds; // Apply yMaxInBounds condition
}