#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yMaxDeltaIntegralKernel(const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMin, const float *xMax, const float *yMax) {

    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread id calculation
    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int &windowIdx = id;

    if (windowIdx < nWindows && x <= h && y <= w) {

        tmpArray += windowIdx * h * w;

        const int xMinInt = (int)ceil(xMin[windowIdx]-1);
        const int xMaxInt = (int)floor(xMax[windowIdx]);
        const int yMaxInt = (int)floor(yMax[windowIdx]);

        float delta = 0;

        // Precompute common index part and re-use
        int hStride = h + 1;
        int wPlus1 = w + 1;
        int xMaxLim = max(0, min(x+xMaxInt, h)) * wPlus1;
        int xMinLim = max(0, min(x+xMinInt, h)) * wPlus1;

        int yStride1 = max(1, min(y+yMaxInt+1, w));
        int yStride0 = max(0, min(y+yMaxInt, w));

        delta += intData[xMaxLim + yStride1];
        delta -= intData[xMaxLim + yStride0];
        delta -= intData[xMinLim + yStride1];
        delta += intData[xMinLim + yStride0];

        delta *= (y+yMaxInt >= 1 && y+yMaxInt < w);
        tmpArray[(x-1)*w + (y-1)] = delta;
    }
}