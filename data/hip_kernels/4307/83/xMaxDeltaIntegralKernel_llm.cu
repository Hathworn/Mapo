#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xMaxDeltaIntegralKernel(const float *intData, const int intDataStrideChannel, float *tmpArray, const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w, const float *xMax, const float *yMin, const float *yMax) {

    int id = NUM_THREADS * blockIdx.x + threadIdx.x;
    tmpArray += id; // tmpArray now points to our output pixel

    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int windowIdx = id % nWindows; id /= nWindows;
    intData += id * intDataStrideChannel;

    const int globalWindowIdx = (id % nInputPlane) * nWindows + windowIdx; id /= nInputPlane;
    const int &batchIdx = id;

    if (batchIdx < batchSize) {
        const int yMinInt = (int)ceil(yMin[globalWindowIdx]-1);
        const int xMaxInt = (int)floor(xMax[globalWindowIdx]);
        const int yMaxInt = (int)floor(yMax[globalWindowIdx]);

        float delta = 0;
        int wx1 = max(0, min(x+xMaxInt+1, h)) * (w+1);
        int wx0 = max(0, min(x+xMaxInt, h)) * (w+1);
        int wyMax = max(0, min(y+yMaxInt, w));
        int wyMin = max(0, min(y+yMinInt, w));

        // Simplified indexing and calculation for delta
        delta += intData[wx1 + wyMax];
        delta -= intData[wx0 + wyMax];
        delta -= intData[wx1 + wyMin];
        delta += intData[wx0 + wyMin];

        delta *= (x+xMaxInt >= 1 && x+xMaxInt < h);
        *tmpArray = delta;
    }
}