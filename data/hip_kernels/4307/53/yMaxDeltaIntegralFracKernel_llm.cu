#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yMaxDeltaIntegralFracKernel( const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMin, const float *xMax, const float *yMax, const float *inData, const int inDataStrideRow) {
    
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread ID calculation
    const int totalSize = h * w;
    
    while (id < nWindows * totalSize) {  // Process more elements per block
        const int windowIdx = id / totalSize;
        int tmpID = id % totalSize;
        const int y = (tmpID % w) + 1;  // 1-indexed
        const int x = (tmpID / w) + 1;  // 1-indexed

        if (x <= h && y <= w) {
            tmpArray += windowIdx * totalSize;

            const int rem = windowIdx % 4;

            if (rem == 3) {
                tmpArray[(x - 1) * w + (y - 1)] = 0;
            } else {
                const float xMinStretched = rem == 0 ? -h : xMin[3 * (windowIdx / 4) + (rem > 0 ? (rem - 1) : rem)];
                const float xMaxStretched = rem == 1 ? h : xMax[3 * (windowIdx / 4) + (rem > 1 ? (rem - 1) : rem)];
                const float yMaxStretched = rem == 3 ? w : yMax[3 * (windowIdx / 4) + (rem > 3 ? (rem - 1) : rem)];

                const int xMinInt = (int)ceil(xMinStretched - 1);
                const float xMinFrac = xMinInt - xMinStretched + 1;

                const int xMaxInt = (int)floor(xMaxStretched);
                const float xMaxFrac = xMaxStretched - xMaxInt;

                const int yMaxInt = (int)floor(yMaxStretched);

                const float trCorner = (y + yMaxInt >= w || x + xMinInt < 1) ? 0 : inData[max(0, min(h - 1, x + xMinInt - 1)) * inDataStrideRow + max(0, min(w - 1, y + yMaxInt))];
                const float brCorner = (y + yMaxInt >= w || x + xMaxInt >= h) ? 0 : inData[max(0, min(h - 1, x + xMaxInt)) * inDataStrideRow + max(0, min(w - 1, y + yMaxInt))];

                float delta = 0;

                delta += trCorner * (x + xMinInt >= h ? 1.0f : xMinFrac);
                delta += brCorner * (x + xMaxInt < 1 ? 1.0f : xMaxFrac);

                delta += intData[max(0, min(x + xMaxInt, h)) * (w + 1) + max(0, min(y + yMaxInt + 1, w))];
                delta -= intData[max(0, min(x + xMaxInt, h)) * (w + 1) + max(0, min(y + yMaxInt, w))];
                delta -= intData[max(0, min(x + xMinInt, h)) * (w + 1) + max(0, min(y + yMaxInt + 1, w))];
                delta += intData[max(0, min(x + xMinInt, h)) * (w + 1) + max(0, min(y + yMaxInt, w))];

                delta *= (y + yMaxInt >= 1 && y + yMaxInt < w);
                tmpArray[(x - 1) * w + (y - 1)] = delta;
            }
        }

        id += gridDim.x * blockDim.x;  // Ensure all threads are used efficiently
    }
}