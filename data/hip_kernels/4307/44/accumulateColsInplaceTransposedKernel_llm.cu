#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulateColsInplaceTransposedKernel(float *input, int channels, int h, int w) {
    // Calculate global column index for this thread
    int colIdx = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;

    // Check if column index is within bounds
    if (colIdx < channels * h) {
        // Zero the (0,0) corner of the output
        input[(colIdx / h) * (h+1)] = 0;

        // Adjust column index for (h+1)-array indexing
        colIdx += colIdx / h + 1;

        // Initialize first element of every column to zero
        input[colIdx] = 0;

        // Use double precision for accumulation to avoid precision loss
        double sum = 0;

        // Loop through elements, accumulate and update values in-place
        for (int i = 1; i <= w; ++i) {
            float *currentElement = &input[i * channels * (h+1) + colIdx];
            sum += static_cast<double>(*currentElement);  // Accumulate sum with double precision
            *currentElement = static_cast<float>(sum);    // Store result back in float
        }
    }
}