#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xMaxDeltaIntegralFracKernel( const float *intData, const int intDataStrideChannel, float *tmpArray, const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w, const float *xMax, const float *yMin, const float *yMax, const float *inData, const int inDataStrideRow, const int inDataStrideChannel) {

    int id = NUM_THREADS * blockIdx.x + threadIdx.x;
    tmpArray += id; // tmpArray now points to our output pixel

    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int windowIdx = id % nWindows; id /= nWindows;

    // `id` is now is now the current global input plane number
    intData  += id * intDataStrideChannel;
    inData   += id *  inDataStrideChannel;

    const int globalWindowIdx = (id % nInputPlane) * nWindows + windowIdx; id /= nInputPlane;
    const int & batchIdx = id;

    if (batchIdx < batchSize) {

        const int yMinInt = __float2int_rz(ceilf(yMin[globalWindowIdx]-1.0f));
        const float yMinFrac = yMinInt - yMin[globalWindowIdx] + 1.0f;

        const int xMaxInt = __float2int_rz(floorf(xMax[globalWindowIdx]));
        const int yMaxInt = __float2int_rz(floorf(yMax[globalWindowIdx]));
        const float yMaxFrac = yMax[globalWindowIdx] - yMaxInt;

        // Consolidate multiple condition checks into a single valid flag
        int xMaxValid = (x + xMaxInt >= 0) & (x + xMaxInt < h);

        int valid = (y + yMinInt >= 1) & (y + yMinInt <= w) & xMaxValid;
        const float blCorner = valid * inData[
            max(0, min(h-1, x+xMaxInt)) * inDataStrideRow +
            max(0, min(w-1, y+yMinInt-1))];

        valid = (y + yMaxInt >= 0) & (y + yMaxInt < w) & xMaxValid;
        const float brCorner = valid * inData[
            max(0, min(h-1, x+xMaxInt)) * inDataStrideRow +
            max(0, min(w-1, y+yMaxInt))];

        float delta = 0;

        delta += brCorner * yMaxFrac;
        delta += blCorner * yMinFrac;

        // Optimize boundary checks by pre-computing indices
        int upperX = max(0, min(x + xMaxInt + 1, h));
        int lowerX = max(0, min(x + xMaxInt, h));
        int upperY = max(0, min(y + yMaxInt, w));
        int lowerY = max(0, min(y + yMinInt, w));

        delta += intData[upperX * (w + 1) + upperY];
        delta -= intData[lowerX * (w + 1) + upperY];
        delta -= intData[upperX * (w + 1) + lowerY];
        delta += intData[lowerX * (w + 1) + lowerY];

        delta *= xMaxValid;
        *tmpArray = delta;
    }
}
```
