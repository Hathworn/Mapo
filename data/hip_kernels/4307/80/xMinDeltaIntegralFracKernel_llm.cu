#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void xMinDeltaIntegralFracKernel(const float *intData, const int intDataStrideChannel, float *tmpArray, const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w, const float *xMin, const float *yMin, const float *yMax, const float *inData, const int inDataStrideRow, const int inDataStrideChannel) {
    int id = NUM_THREADS * blockIdx.x + threadIdx.x;
    tmpArray += id; // tmpArray now points to our output pixel

    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int windowIdx = id % nWindows; id /= nWindows;

    // `id` is now the current global input plane number
    intData  += id * intDataStrideChannel;
    inData += id * inDataStrideChannel;

    const int globalWindowIdx = (id % nInputPlane) * nWindows + windowIdx; id /= nInputPlane;
    const int & batchIdx = id;

    if (batchIdx < batchSize) {
        const int xMinInt = (int)ceil(xMin[globalWindowIdx]-1);

        const int yMinInt = (int)ceil(yMin[globalWindowIdx]-1);
        const float yMinFrac = yMinInt-yMin[globalWindowIdx]+1;

        const int yMaxInt = (int)floor(yMax[globalWindowIdx]);
        const float yMaxFrac = yMax[globalWindowIdx]-yMaxInt;

        bool valid = (y+yMinInt >= 1) & (y+yMinInt <= w) & (x+xMinInt >= 1);
        
        // Optimize memory access by avoiding repeated calculations
        int validYMin = max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow + max(0,min(w-1,y+yMinInt-1));
        int validYMax = max(0,min(h-1,x+xMinInt-1)) * inDataStrideRow + max(0,min(w-1,y+yMaxInt));

        const float tlCorner = valid * inData[validYMin];
        const float trCorner = valid & (y+yMaxInt >= 0) & (y+yMaxInt < w) ? inData[validYMax] : 0;
        
        float delta = 0;

        delta += trCorner * yMaxFrac;
        delta += tlCorner * yMinFrac;
        
        // Reduce redundant calculations by storing indices
        int idx1 = max(0,min(x+xMinInt  , h))*(w+1) + max(0,min(y+yMaxInt  , w));
        int idx2 = max(0,min(x+xMinInt-1, h))*(w+1) + max(0,min(y+yMaxInt  , w));
        int idx3 = max(0,min(x+xMinInt  , h))*(w+1) + max(0,min(y+yMinInt  , w));
        int idx4 = max(0,min(x+xMinInt-1, h))*(w+1) + max(0,min(y+yMinInt  , w));

        delta += intData[idx1];
        delta -= intData[idx2];
        delta -= intData[idx3];
        delta += intData[idx4];

        delta *= (x+xMinInt >= 1) & (x+xMinInt <= h);
        *tmpArray = -delta;
    }
}