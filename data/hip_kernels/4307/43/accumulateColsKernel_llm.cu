#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulateColsKernel(float *input, float *output, int channels, int h, int w) {
    // compute global column index
    int colIdx = blockDim.x * blockIdx.x + threadIdx.x;

    if (colIdx < channels * w) {
        // compute channel and its local column index
        int channel = colIdx / w;
        int localColIdx = colIdx % w + 1; // 1-indexed
    
        // adjust pointers to the start of the current channel
        input  += channel * h * w;
        output += channel * (h+1) * (w+1);

        output[localColIdx] = 0; // first element of every column is always zero
        double sum = 0.0;

        for (int i = 1; i <= h; ++i) {
            sum += static_cast<double>(input[(i-1) * w + localColIdx - 1]);
            output[i * (w+1) + localColIdx] = static_cast<float>(sum);
        }
    }
}