#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dirtyFixWindowsVarScaleKernel(float *xMin, float *xMax, float *yMin, float *yMax, const int size, const float h, const float w, const float minWidth) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplify indexing calculation

    if (idx < 2 * size) {
        float paramMin, paramMax;

        bool isX = (idx < size); // Determine whether this thread works on x or y

        // Compute correct index for y
        int effectiveIdx = isX ? idx : idx - size;

        // Select appropriate bounds based on x or y
        float boundMin = isX ? -h + 1 : -w + 1;
        float boundMax = isX ? h - 1 : w - 1;
        float* arrayMin = isX ? xMin : yMin;
        float* arrayMax = isX ? xMax : yMax;

        // Calculate parameter min/max using selected bounds
        paramMin = max(boundMin, min(boundMax, arrayMin[effectiveIdx]));
        paramMax = max(boundMin, min(boundMax, arrayMax[effectiveIdx]));

        // Adjust parameters if necessary
        if (paramMin + minWidth - 0.99 > paramMax) {
            const float mean = 0.5f * (paramMin + paramMax);
            paramMin = mean - 0.5f * (minWidth - 0.9f);
            paramMax = mean + 0.5f * (minWidth - 0.9f);
        }

        // Save results to correct arrays
        arrayMin[effectiveIdx] = paramMin;
        arrayMax[effectiveIdx] = paramMax;
    }
}