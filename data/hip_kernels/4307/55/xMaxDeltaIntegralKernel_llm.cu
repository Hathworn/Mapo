#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xMaxDeltaIntegralKernel(const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMax, const float *yMin, const float *yMax) {

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int& windowIdx = id;

    if (windowIdx < nWindows) {
        tmpArray += windowIdx * h * w;

        const int yMinInt = (int)ceil(yMin[windowIdx] - 1);
        const int xMaxInt = (int)floor(xMax[windowIdx]);
        const int yMaxInt = (int)floor(yMax[windowIdx]);

        float delta = 0;

        // Use conditional operator to reduce branching
        int xOffsetMax = max(1, min(x + xMaxInt + 1, h));
        int xOffset = max(0, min(x + xMaxInt, h));
        int yOffsetMax = max(0, min(y + yMaxInt, w));
        int yOffsetMin = max(0, min(y + yMinInt, w));

        // Reduce redundant calls to intData
        delta += intData[xOffsetMax * (w + 1) + yOffsetMax];
        delta -= intData[xOffset * (w + 1) + yOffsetMax];
        delta -= intData[xOffsetMax * (w + 1) + yOffsetMin];
        delta += intData[xOffset * (w + 1) + yOffsetMin];

        delta *= (x + xMaxInt >= 1 && x + xMaxInt < h);
        tmpArray[(x - 1) * w + (y - 1)] = delta;
    }
}