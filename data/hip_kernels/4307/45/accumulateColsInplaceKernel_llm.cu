#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulateColsInplaceKernel(float *input, int channels, int h, int w) {
    // Optimization: Use shared memory to store partial sums
    extern __shared__ double sharedSum[];

    // global column index (of all `channels * w` columns in this image)
    int colIdx = blockDim.x * blockIdx.x + threadIdx.x;

    if (colIdx < channels * w) {
        input += (colIdx / w) * (h + 1) * (w + 1); // jump to current channel
        colIdx %= w; // switch to local column index,
        ++colIdx;    // it's 1-indexed because first output column is always zero

        input[colIdx] = 0; // first element of every column is always zero
        double sum = 0;

        for (int i = 1; i <= h; ++i) {
            float *currentElement = &input[i * (w + 1) + colIdx];
            sum += static_cast<double>(*currentElement);
            sharedSum[threadIdx.x] = sum; // Store sum in shared memory
            __syncthreads(); // Ensure all threads have finished updating shared memory
            *currentElement = static_cast<float>(sharedSum[threadIdx.x]); // Use value from shared memory
        }
    }
}