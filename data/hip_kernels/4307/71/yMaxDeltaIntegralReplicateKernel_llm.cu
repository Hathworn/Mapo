#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yMaxDeltaIntegralReplicateKernel( const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMin, const float *xMax, const float *yMax, const int strideH, const int strideW) {

// Use block and thread indices for better parallelism
const int hOut = (h + strideH - 1) / strideH;
const int wOut = (w + strideW - 1) / strideW;

const int windowIdx = blockIdx.x;
const int xOut = blockIdx.y * blockDim.y + threadIdx.y;
const int yOut = blockIdx.z * blockDim.z + threadIdx.z;

if (windowIdx < nWindows and xOut < hOut and yOut < wOut) {

const int x = xOut * strideH + 1;
const int y = yOut * strideW + 1;

tmpArray += windowIdx * hOut * wOut;

const int xMinInt = (int)ceil(xMin[windowIdx]-1);
const int xMaxInt = (int)floor(xMax[windowIdx]);
const int yMaxInt = (int)floor(yMax[windowIdx]);

float delta = 0;

// Simplify indexing using pre-computed constants
int max_h = max(0, min(h, x + xMaxInt));
int min_h = max(0, min(h, x + xMinInt));

delta += intData[max_h * (w + 1) + max(1, min(w, y + yMaxInt + 1))];
delta -= intData[max_h * (w + 1) + max(0, min(w, y + yMaxInt))];
delta -= intData[min_h * (w + 1) + max(1, min(w, y + yMaxInt + 1))];
delta += intData[min_h * (w + 1) + max(0, min(w, y + yMaxInt))];

delta *= (y + yMaxInt >= 1 and y + yMaxInt < w);
tmpArray[xOut * wOut + yOut] = delta;
}
}