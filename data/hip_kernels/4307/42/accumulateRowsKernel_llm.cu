#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulateRowsKernel(float *input, float *output, int channels, int h, int w) {
    int globalRowIdx = blockIdx.x * blockDim.x + threadIdx.x; // Simplified block index calculation

    if (globalRowIdx < channels * h) {
        float *outputRow = output + (globalRowIdx + globalRowIdx / h + 1) * (w + 1) + 1;
        outputRow[-1] = 0;

        // Using shared memory to enhance memory access pattern
        extern __shared__ float sharedSum[];
        double sum = 0.0;

        for (int i = 0; i < w; ++i) {
            sum += input[globalRowIdx * w + i];
            sharedSum[threadIdx.x] = sum; // Store intermediate sum in shared memory
            __syncthreads(); // Ensure all threads have written the shared memory

            outputRow[i] = static_cast<float>(sharedSum[threadIdx.x]); // Write from shared memory to global
        }

        // Zero the (0,0) corner of the output separately
        if (threadIdx.x == 0) {
            output[(globalRowIdx / h) * (w + 1) * (h + 1)] = 0;
        }
    }
}