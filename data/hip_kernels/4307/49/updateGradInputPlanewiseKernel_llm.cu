#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateGradInputPlanewiseKernel( float *gradOutputIntData, float *gradInputData, int h, int w, int nWindows, float *xMin, float *xMax, float *yMin, float *yMax) {

    const int x = BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = BLOCK_SIZE * blockIdx.y + threadIdx.y;

    if (x < h && y < w) {
        double outValue = 0;

        for (int windowIdx = 0; windowIdx < nWindows; ++windowIdx) {

            int xMinCurr = (int)ceil(-xMax[windowIdx]);
            int yMinCurr = (int)ceil(-yMax[windowIdx]);
            int xMaxCurr = (int)floor(-xMin[windowIdx]) + 1;
            int yMaxCurr = (int)floor(-yMin[windowIdx]) + 1;

            // Simplified boundary conditions using ternary operators
            xMinCurr = (x == 0 && xMaxCurr >= 0) ? 0 : xMinCurr;
            xMaxCurr = (x == h-1 && xMinCurr <= 0) ? h+66 : xMaxCurr;
            yMinCurr = (y == 0 && yMaxCurr >= 0) ? 0 : yMinCurr;
            yMaxCurr = (y == w-1 && yMinCurr <= 0) ? w+66 : yMaxCurr;

            const int t = max(0, min(x + xMinCurr, h));
            const int b = max(0, min(x + xMaxCurr, h));
            const int l = max(0, min(y + yMinCurr, w));
            const int r = max(0, min(y + yMaxCurr, w));

            // Unroll calculations for better performance
            outValue += gradOutputIntData[b * (w + 1) + r] - gradOutputIntData[t * (w + 1) + r]
                        - gradOutputIntData[b * (w + 1) + l] + gradOutputIntData[t * (w + 1) + l];

            // Move to the next channel
            gradOutputIntData += (h + 1) * (w + 1);
        }

        gradInputData[x * w + y] = outValue;
    }
}