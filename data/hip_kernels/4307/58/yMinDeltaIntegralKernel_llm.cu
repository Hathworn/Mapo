#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yMinDeltaIntegralKernel(const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMin, const float *xMax, const float *yMin) {

    int id = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows && x <= h && y <= w) {

        tmpArray += windowIdx * h * w;
        
        const int xMinInt = static_cast<int>(ceilf(xMin[windowIdx] - 1));
        const int yMinInt = static_cast<int>(ceilf(yMin[windowIdx] - 1));
        const int xMaxInt = static_cast<int>(floorf(xMax[windowIdx]));
        
        float delta = 0;

        // Pre-compute common index offsets to reduce redundant calculations
        int offset1 = max(0, min(x + xMaxInt, h)) * (w + 1);
        int offset2 = max(0, min(x + xMinInt, h)) * (w + 1);
        int yOffset1 = max(0, min(y + yMinInt, w));
        int yOffset2 = max(0, min(y + yMinInt - 1, w - 1));

        delta += intData[offset1 + yOffset1];
        delta -= intData[offset1 + yOffset2];
        delta -= intData[offset2 + yOffset1];
        delta += intData[offset2 + yOffset2];

        // Apply condition only once
        if (y + yMinInt >= 1 && y + yMinInt < w) {
            tmpArray[(x - 1) * w + (y - 1)] *= -delta;
        }
    }
}