#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateGradInputVarScaleKernel(float *gradOutputIntData, float *gradInputData, int h, int w, int nWindows, float *xMin, float *xMax, float *yMin, float *yMax) {

    // Compute flattened thread identifier for better coalescing    
    const int tid = blockIdx.y * blockDim.y * w + blockIdx.x * blockDim.x + threadIdx.y * w + threadIdx.x;
    const int x = tid / w;
    const int y = tid % w;

    if (x < h && y < w) {
        double outValue = 0;
        
        for (int windowIdx = 0; windowIdx < nWindows; ++windowIdx) {
            int xMinCurr = (int)ceil(-xMax[windowIdx]);
            int yMinCurr = (int)ceil(-yMax[windowIdx]);
            int xMaxCurr = (int)floor(-xMin[windowIdx]) + 1;
            int yMaxCurr = (int)floor(-yMin[windowIdx]) + 1;

            // Simplified conditional checks using ternary operators
            xMinCurr = (x == 0 && xMaxCurr >= 0) ? 0 : xMinCurr;
            xMaxCurr = (x == h - 1 && xMinCurr <= 0) ? h + 66 : xMaxCurr;
            yMinCurr = (y == 0 && yMaxCurr >= 0) ? 0 : yMinCurr;
            yMaxCurr = (y == w - 1 && yMinCurr <= 0) ? w + 66 : yMaxCurr;

            const int t = max(0, min(x + xMinCurr, h));
            const int b = max(0, min(x + xMaxCurr, h));
            const int l = max(0, min(y + yMinCurr, w));
            const int r = max(0, min(y + yMaxCurr, w));

            outValue += gradOutputIntData[b * (w + 1) + r];
            outValue -= gradOutputIntData[t * (w + 1) + r];
            outValue -= gradOutputIntData[b * (w + 1) + l];
            outValue += gradOutputIntData[t * (w + 1) + l];

            // Move to the next channel
            gradOutputIntData += (h + 1) * (w + 1);
        }

        gradInputData[x * w + y] = outValue;
    }
}