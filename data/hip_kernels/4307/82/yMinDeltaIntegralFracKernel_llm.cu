#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yMinDeltaIntegralFracKernel(const float *intData, const int intDataStrideChannel, float *tmpArray, const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w, const float *xMin, const float *xMax, const float *yMin, const float *inData, const int inDataStrideRow, const int inDataStrideChannel) {

    int id = NUM_THREADS * blockIdx.x + threadIdx.x;
    tmpArray += id; // tmpArray now points to our output pixel

    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int windowIdx = id % nWindows; id /= nWindows;

    intData += id * intDataStrideChannel;
    inData += id * inDataStrideChannel;

    const int globalWindowIdx = (id % nInputPlane) * nWindows + windowIdx; id /= nInputPlane;
    const int & batchIdx = id;

    if (batchIdx < batchSize) {
        const int xMinInt = (int)ceil(xMin[globalWindowIdx] - 1);
        const float xMinFrac = xMinInt - xMin[globalWindowIdx] + 1;

        const int yMinInt = (int)ceil(yMin[globalWindowIdx] - 1);

        const int xMaxInt = (int)floor(xMax[globalWindowIdx]);
        const float xMaxFrac = xMax[globalWindowIdx] - xMaxInt;

        int valid;

        // Optimize bounds checking and indexing
        valid = not (y + yMinInt < 1) & not (x + xMinInt < 1) & not (x + xMinInt > h);
        const int tlIndexRow = max(0, min(h - 1, x + xMinInt - 1));
        const int tlIndexCol = max(0, min(w - 1, y + yMinInt - 1));
        const float tlCorner = valid * inData[tlIndexRow * inDataStrideRow + tlIndexCol];

        valid = not (y + yMinInt < 1) & not (x + xMaxInt < 0) & not (x + xMaxInt >= h);
        const int blIndexRow = max(0, min(h - 1, x + xMaxInt));
        const int blIndexCol = tlIndexCol; // Same col as tlCorner
        const float blCorner = valid * inData[blIndexRow * inDataStrideRow + blIndexCol];

        // Reduce redundant calculations
        const int intDataRow_xMax = max(0, min(x + xMaxInt, h)) * (w + 1);
        const int intDataRow_xMin = max(0, min(x + xMinInt, h)) * (w + 1);

        float delta = 0;
        delta += tlCorner * xMinFrac;
        delta += blCorner * xMaxFrac;

        delta += intData[intDataRow_xMax + max(0, min(y + yMinInt, w))];
        delta -= intData[intDataRow_xMax + max(0, min(y + yMinInt - 1, w))];
        delta -= intData[intDataRow_xMin + max(0, min(y + yMinInt, w))];
        delta += intData[intDataRow_xMin + max(0, min(y + yMinInt - 1, w))];

        delta *= (y + yMinInt >= 1) & (y + yMinInt <= w);
        *tmpArray = -delta;
    }
}