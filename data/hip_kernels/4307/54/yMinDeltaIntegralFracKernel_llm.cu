#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yMinDeltaIntegralFracKernel(const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMin, const float *xMax, const float *yMin, const float *inData, const int inDataStrideRow) {

    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows and x <= h and y <= w) {

        tmpArray += windowIdx * h * w;

        const int rem = windowIdx % 4;

        if (rem == 2) {
            tmpArray[(x-1)*w + (y-1)] = 0;
        } else {

            const float xMinStretched = rem == 0 ? -h :
                                        xMin[3*(windowIdx/4) + (rem > 0 ? (rem-1) : rem)];
            const float xMaxStretched = rem == 1 ?  h :
                                        xMax[3*(windowIdx/4) + (rem > 1 ? (rem-1) : rem)];
            const float yMinStretched = rem == 2 ? -w :
                                        yMin[3*(windowIdx/4) + (rem > 2 ? (rem-1) : rem)];

            // Use registers for frequently accessed variables
            const int xMinInt = (int)ceil(xMinStretched-1);
            const float xMinFrac = xMinInt-xMinStretched+1;

            const int yMinInt = (int)ceil(yMinStretched-1);

            const int xMaxInt = (int)floor(xMaxStretched);
            const float xMaxFrac = xMaxStretched-xMaxInt;

            // Simplify boundary access calculations
            const int xMinIndex = max(0, min(h-1, x+xMinInt-1)) * inDataStrideRow;
            const int xMaxIndex = max(0, min(h-1, x+xMaxInt  )) * inDataStrideRow;
            const int yMinIndex = max(0, min(w-1, y+yMinInt-1));

            const float tlCorner = (y+yMinInt < 1 || x+xMinInt < 1) ? 
                                    0 : inData[xMinIndex + yMinIndex];
            const float blCorner = (y+yMinInt < 1 || x+xMaxInt >= h) ? 
                                    0 : inData[xMaxIndex + yMinIndex];

            float delta = 0;

            delta += tlCorner * (x+xMinInt >= h ? 1.0f : xMinFrac);
            delta += blCorner * (x+xMaxInt <  1 ? 1.0f : xMaxFrac);

            // Reduce repeated computation
            const int xMax1 = max(0, min(x+xMaxInt, h));
            const int xMin1 = max(0, min(x+xMinInt, h));
            const int yMin1 = max(0, min(y+yMinInt, w));
            const int yMin0 = max(0, min(y+yMinInt-1, w));

            delta += intData[xMax1*(w+1) + yMin1];
            delta -= intData[xMax1*(w+1) + yMin0];
            delta -= intData[xMin1*(w+1) + yMin1];
            delta += intData[xMin1*(w+1) + yMin0];

            delta *= (y+yMinInt >= 1 and y+yMinInt < w);
            tmpArray[(x-1)*w + (y-1)] *= -delta;
        }
    }
}