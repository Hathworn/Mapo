#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xMinDeltaIntegralKernel( const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMin, const float *yMin, const float *yMax) {

    int id = BLOCK_SIZE * BLOCK_SIZE * blockIdx.x + threadIdx.x;
    const int y = id % w + 1; id /= w; // 1-indexed
    const int x = id % h + 1; id /= h; // 1-indexed
    const int & windowIdx = id;

    if (windowIdx < nWindows && x <= h && y <= w) {

        tmpArray += windowIdx * h * w;

        const int xMinInt = (int)ceil(xMin[windowIdx] - 1);
        const int yMinInt = (int)ceil(yMin[windowIdx] - 1);
        const int yMaxInt = (int)floor(yMax[windowIdx]);

        float delta = 0;

        // Optimize memory access by reducing duplicated calculations
        const int xIndexPos = max(0, min(x + xMinInt, h - 1)) * (w + 1);
        const int xIndexNeg = max(0, min(x + xMinInt - 1, h)) * (w + 1);
        const int yIndexMax = max(0, min(y + yMaxInt, w));
        const int yIndexMin = max(0, min(y + yMinInt, w));

        delta += intData[xIndexPos + yIndexMax];
        delta -= intData[xIndexNeg + yIndexMax];
        delta -= intData[xIndexPos + yIndexMin];
        delta += intData[xIndexNeg + yIndexMin];

        if(x + xMinInt >= 1 && x + xMinInt < h) {
            tmpArray[(x-1) * w + (y-1)] *= -delta;  // Combine condition check with array update
        }
    }
}