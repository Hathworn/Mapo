#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yMinDeltaIntegralReplicateFracKernel(const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMin, const float *xMax, const float *yMin, const float *inData, const int inDataStrideRow, const int strideH, const int strideW) {

    // Calculate hOut and wOut once
    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;

    // Use blockDim and gridDim for id calculation
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    const int totalThreads = gridDim.x * blockDim.x; // Total number of threads

    // Iterate over all elements with stride equal to totalThreads
    for (; id < nWindows * hOut * wOut; id += totalThreads) {
        const int yOut = id % wOut; id /= wOut; // 0-indexed
        const int xOut = id % hOut; id /= hOut; // 0-indexed
        const int & windowIdx = id;

        if (windowIdx < nWindows) {

            const int x = xOut * strideH + 1;
            const int y = yOut * strideW + 1;

            float *currentTmpArray = tmpArray + windowIdx * hOut * wOut;

            const int xMinInt = static_cast<int>(ceil(xMin[windowIdx] - 1));
            const float xMinFrac = xMinInt - xMin[windowIdx] + 1;

            const int yMinInt = static_cast<int>(ceil(yMin[windowIdx] - 1));

            const int xMaxInt = static_cast<int>(floor(xMax[windowIdx]));
            const float xMaxFrac = xMax[windowIdx] - xMaxInt;

            const float tlCorner = y + yMinInt <  1 || x + xMinInt <  1 ? 
                0 : inData[max(0, min(h-1, x + xMinInt - 1)) * inDataStrideRow + max(0, min(w-1, y + yMinInt - 1))];
            const float blCorner = y + yMinInt <  1 || x + xMaxInt >= h ? 
                0 : inData[max(0, min(h-1, x + xMaxInt)) * inDataStrideRow + max(0, min(w-1, y + yMinInt - 1))];

            float delta = 0;

            delta += tlCorner * (x + xMinInt >= h ? 1.0f : xMinFrac);
            delta += blCorner * (x + xMaxInt <  1 ? 1.0f : xMaxFrac);

            delta += intData[max(0, min(x + xMaxInt, h)) * (w+1) + max(0, min(y + yMinInt, w))];
            delta -= intData[max(0, min(x + xMaxInt, h)) * (w+1) + max(0, min(y + yMinInt - 1, w))];
            delta -= intData[max(0, min(x + xMinInt, h)) * (w+1) + max(0, min(y + yMinInt, w))];
            delta += intData[max(0, min(x + xMinInt, h)) * (w+1) + max(0, min(y + yMinInt - 1, w))];

            delta *= (y + yMinInt >= 1 && y + yMinInt < w);
            currentTmpArray[xOut * wOut + yOut] *= -delta;
        }
    }
}