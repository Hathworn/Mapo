#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void xMinDeltaIntegralKernel(const float *intData, const int intDataStrideChannel, float *tmpArray, const int batchSize, const int nInputPlane, const int nWindows, const int h, const int w, const float *xMin, const float *yMin, const float *yMax) {

    // Calculate global thread ID
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Precompute offsets for input data
    tmpArray += id;
    const int y = id % w + 1; id /= w;
    const int x = id % h + 1; id /= h;
    const int windowIdx = id % nWindows; id /= nWindows;
    intData += id * intDataStrideChannel;

    const int globalWindowIdx = (id % nInputPlane) * nWindows + windowIdx; id /= nInputPlane;
    const int batchIdx = id;

    // Check if batch index is within bounds
    if (batchIdx < batchSize) {

        // Precompute coordinates rounded
        const int xMinInt = (int)ceil(xMin[globalWindowIdx] - 1);
        const int yMinInt = (int)ceil(yMin[globalWindowIdx] - 1);
        const int yMaxInt = (int)floor(yMax[globalWindowIdx]);

        // Initialize delta to zero
        float delta = 0;

        // Simplify access with direct index calculation
        int xOffset = max(0, min(x + xMinInt, h)) * (w + 1);

        delta += intData[xOffset + max(0, min(y + yMaxInt, w))];
        delta -= intData[xOffset + max(0, min(y + yMinInt, w))];
        
        xOffset = max(0, min(x + xMinInt - 1, h)) * (w + 1);
        
        delta -= intData[xOffset + max(0, min(y + yMaxInt, w))];
        delta += intData[xOffset + max(0, min(y + yMinInt, w))];

        // Apply boundary condition check
        delta *= (x + xMinInt >= 1 && x + xMinInt < h);

        // Store result in output array
        *tmpArray = -delta;
    }
}