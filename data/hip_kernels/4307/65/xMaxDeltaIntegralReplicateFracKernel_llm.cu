#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void xMaxDeltaIntegralReplicateFracKernel(const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMax, const float *yMin, const float *yMax, const float *inData, const int inDataStrideRow, const int strideH, const int strideW) {
    // Define hOut and wOut using blockDim.x for better thread utilization
    const int hOut = (h + strideH - 1) / strideH;
    const int wOut = (w + strideW - 1) / strideW;

    int id = blockDim.x * blockDim.y * blockIdx.x + threadIdx.x;
    const int yOut = id % wOut; id /= wOut; // 0-indexed
    const int xOut = id % hOut; id /= hOut; // 0-indexed
    const int &windowIdx = id;

    if (windowIdx < nWindows && xOut < hOut && yOut < wOut) {
        const int x = xOut * strideH + 1;
        const int y = yOut * strideW + 1;

        tmpArray += windowIdx * hOut * wOut;

        const int yMinInt = (int)ceilf(yMin[windowIdx] - 1);
        const float yMinFrac = yMinInt - yMin[windowIdx] + 1;

        const int xMaxInt = (int)floorf(xMax[windowIdx]);

        const int yMaxInt = (int)floorf(yMax[windowIdx]);
        const float yMaxFrac = yMax[windowIdx] - yMaxInt;

        const float blCorner = (y + yMinInt < 1 || x + xMaxInt >= h) ? 0 : inData[max(0, min(h - 1, x + xMaxInt)) * inDataStrideRow + max(0, min(w - 1, y + yMinInt - 1))];
        const float brCorner = (y + yMaxInt >= w || x + xMaxInt >= h) ? 0 : inData[max(0, min(h - 1, x + xMaxInt)) * inDataStrideRow + max(0, min(w - 1, y + yMaxInt))];

        float delta = 0;

        delta += brCorner * ((y + yMaxInt < 1) ? 1.0f : yMaxFrac);
        delta += blCorner * ((y + yMinInt >= w) ? 1.0f : yMinFrac);

        delta += intData[max(0, min(x + xMaxInt + 1, h)) * (w + 1) + max(0, min(y + yMaxInt, w))];
        delta -= intData[max(0, min(x + xMaxInt, h)) * (w + 1) + max(0, min(y + yMaxInt, w))];
        delta -= intData[max(0, min(x + xMaxInt + 1, h)) * (w + 1) + max(0, min(y + yMinInt, w))];
        delta += intData[max(0, min(x + xMaxInt, h)) * (w + 1) + max(0, min(y + yMinInt, w))];

        delta *= (x + xMaxInt >= 1 && x + xMaxInt < h);
        tmpArray[xOut * wOut + yOut] = delta;
    }
}