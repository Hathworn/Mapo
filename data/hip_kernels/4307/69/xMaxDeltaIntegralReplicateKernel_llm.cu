#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xMaxDeltaIntegralReplicateKernel( const float *intData, float *tmpArray, const int nWindows, const int h, const int w, const float *xMax, const float *yMin, const float *yMax, const int strideH, const int strideW) {

// Calculate output dimensions
const int hOut = (h + strideH - 1) / strideH;
const int wOut = (w + strideW - 1) / strideW;

// Use 2D blockIdx and 2D threadIdx for better parallelization
int id = blockIdx.x * blockDim.x + threadIdx.x;
if (id >= nWindows * hOut * wOut) return; // Check bounds

const int yOut = id % wOut;
id /= wOut;
const int xOut = id % hOut;
id /= hOut;
const int windowIdx = id;

if (windowIdx < nWindows) {
    const int x = xOut * strideH + 1;
    const int y = yOut * strideW + 1;

    tmpArray += windowIdx * hOut * wOut;

    const int yMinInt = (int)ceil(yMin[windowIdx] - 1);
    const int xMaxInt = (int)floor(xMax[windowIdx]);
    const int yMaxInt = (int)floor(yMax[windowIdx]);

    float delta = 0;

    delta += intData[max(1, min(x + xMaxInt + 1, h)) * (w + 1) + max(0, min(y + yMaxInt, w))];
    delta -= intData[max(0, min(x + xMaxInt, h)) * (w + 1) + max(0, min(y + yMaxInt, w))];
    delta -= intData[max(1, min(x + xMaxInt + 1, h)) * (w + 1) + max(0, min(y + yMinInt, w))];
    delta += intData[max(0, min(x + xMaxInt, h)) * (w + 1) + max(0, min(y + yMinInt, w))];

    delta *= (x + xMaxInt >= 1 && x + xMaxInt < h);
    tmpArray[xOut * wOut + yOut] = delta;
}
}