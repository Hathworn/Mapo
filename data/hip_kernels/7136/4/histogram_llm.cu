#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogram(int *hist_out, unsigned char *img_in, int img_w, int img_h, int nbr_bin) {
    // Using shared memory for temporary storage
    extern __shared__ int shared_hist[];
    
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    unsigned int col = tx + blockDim.x * bx;
    unsigned int row = ty + blockDim.y * by;
    int grid_width = gridDim.x * blockDim.x;
    int id = row * grid_width + col;

    // Initialize shared memory
    if (id < nbr_bin)
        shared_hist[id] = 0;

    __syncthreads();

    if (row < img_h && col < img_w) {
        atomicAdd(&(shared_hist[img_in[id]]), 1);
    }

    __syncthreads();

    // Write results back to global memory
    if (id < nbr_bin) {
        atomicAdd(&(hist_out[id]), shared_hist[id]);
    }
}