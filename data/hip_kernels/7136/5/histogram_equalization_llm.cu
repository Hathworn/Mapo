#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogram_equalization(int *lut, unsigned char *img_out, unsigned char *img_in, int img_size) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check within image bound
    if(id < img_size) {
        // Efficient condition check and output assignment
        unsigned char value = lut[img_in[id]] > 255 ? 255 : (unsigned char)lut[img_in[id]];
        img_out[id] = value;
    }
}