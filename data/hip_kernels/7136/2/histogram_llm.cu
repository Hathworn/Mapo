#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histogram( int * hist_out, unsigned char * img_in, int img_w,int img_h,  int nbr_bin){

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    __shared__ int smem[256];
    
    // Initialize shared memory array using one thread
    if (tx == 0 && ty == 0) {
        for (int i = 0; i < 256; i++) {
            smem[i] = 0;
        }
    }
    __syncthreads();

    unsigned int col = tx + blockDim.x * bx;
    unsigned int row = ty + blockDim.y * by;

    int grid_width = gridDim.x * blockDim.x;
    int id = row * grid_width + col;

    // Ensure threads go out of bounds
    if (row < img_w && col < img_h) {
        atomicAdd(&(smem[img_in[id]]), 1);
    }

    __syncthreads();

    // Atomic add to global histogram using single block thread
    if (tx == 0 && ty == 0) {
        for (int i = 0; i < 256; i++) {
            atomicAdd(&(hist_out[i]), smem[i]);
        }
    }
}