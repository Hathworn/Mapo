#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ConvolutionColGPU(float *d_Dst, float *d_Src, float *d_Filter, int imageW, int imageH, int filterR) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (col < imageW && row < imageH) { // Boundary check for active threads
        float sum = 0;
        for (int k = -filterR; k <= filterR; k++) {
            int d = row + k;
            if (d >= 0 && d < imageH) { // Correct boundary condition
                sum += d_Src[col + imageW * d] * d_Filter[filterR - k];
            }
        }
        d_Dst[row * imageW + col] = sum;
    }
}