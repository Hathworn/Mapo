#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolutionColumnGPU(float *d_Dst, float *d_Src, float *d_Filter, int imageW, int imageH, int filterR) {
    // Shared memory to store source data
    extern __shared__ float s_Src[];

    // Calculate new image width with padding
    int newImageW = imageW + filterR * 2;

    // Compute global row and col index
    int globalRow = blockDim.y * blockIdx.y + threadIdx.y;
    int globalCol = blockDim.x * blockIdx.x + threadIdx.x;

    // Load data into shared memory with halo region
    int sharedRow = threadIdx.y + filterR;
    int sharedCol = threadIdx.x + filterR;
    s_Src[sharedRow * blockDim.x + sharedCol] = d_Src[(globalRow + filterR) * newImageW + globalCol + filterR];

    // Load halo above
    if (threadIdx.y < filterR) {
        s_Src[threadIdx.y * blockDim.x + sharedCol] = d_Src[(globalRow + threadIdx.y) * newImageW + globalCol + filterR];
    }

    // Load halo below
    if (threadIdx.y >= blockDim.y - filterR) {
        s_Src[(threadIdx.y + 2 * filterR) * blockDim.x + sharedCol] = d_Src[(globalRow + 2 * filterR + threadIdx.y) * newImageW + globalCol + filterR];
    }

    // Synchronize to ensure all loads are complete
    __syncthreads();

    // Perform convolution
    float sum = 0;
    for (int k = -filterR; k <= filterR; k++) {
        sum += s_Src[(sharedRow + k) * blockDim.x + sharedCol] * d_Filter[filterR - k];
    }

    // Write result to global memory
    d_Dst[(globalRow + filterR) * newImageW + globalCol + filterR] = sum;
}