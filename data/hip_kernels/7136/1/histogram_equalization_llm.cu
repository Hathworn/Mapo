#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogram_equalization(int *lut, unsigned char *img_out, unsigned char *img_in, int *hist_in, int img_size, int nbr_bin) {
    // Optimize thread index calculations by using predefined variables
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int ty = threadIdx.y + blockIdx.y * blockDim.y;
    int id = ty * blockDim.x * gridDim.x + tx;

    // Check if the thread ID is within the image size bounds
    if (id < img_size) {
        // Fetch lut value and clamp to 255 if necessary
        int lut_value = lut[img_in[id]];
        img_out[id] = (unsigned char)(lut_value > 255 ? 255 : lut_value);
    }
}