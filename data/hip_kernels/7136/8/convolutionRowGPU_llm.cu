#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolutionRowGPU(float *d_Dst, float *d_Src, float *d_Filter, int imageW, int imageH, int filterR) {
    // Calculate global thread position for row and column
    int row = blockDim.y * blockIdx.y + threadIdx.y + filterR;
    int col = blockDim.x * blockIdx.x + threadIdx.x + filterR;
    int newImageW = imageW + filterR * 2;

    float sum = 0.0f;

    // Unrolling the loop to improve performance
    #pragma unroll
    for (int k = -filterR; k <= filterR; k++) {
        int d = col + k;
        sum += d_Src[row * newImageW + d] * d_Filter[filterR - k];
    }

    // Write the result to global memory
    d_Dst[row * newImageW + col] = sum;
}