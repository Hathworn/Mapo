#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ConvolutionRowGPU(float *d_Dst, float *d_Src, float *d_Filter, int imageW, int imageH, int filterR) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    float sum = 0;

    // Loop unrolling optimization
    #pragma unroll
    for (int k = -filterR; k <= filterR; k++) {
        int d = col + k;
        if (d >= 0 && d < imageW) {
            sum += d_Src[row * imageW + d] * d_Filter[filterR - k];
        }
    }

    // Ensure that the calculated value is inside the image boundary
    if (col < imageW && row < imageH) {
        d_Dst[row * imageW + col] = sum;
    }
}