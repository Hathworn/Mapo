#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogram(int *hist_out, unsigned char *img_in, int img_w, int img_h, int nbr_bin) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    __shared__ int smem[256];
    if(tx < 256)  // Ensure thread does not access out-of-bounds shared memory
        smem[tx] = 0;
    __syncthreads();

    unsigned int col = tx + blockDim.x * bx;
    unsigned int row = ty + blockDim.y * by;

    int grid_width = gridDim.x * blockDim.x;
    int id = row * grid_width + col;

    if (row < img_h && col < img_w)  // Correct condition check for image bounds
        atomicAdd(&(smem[img_in[id]]), 1);

    __syncthreads();

    if(tx < nbr_bin)  // Avoid unnecessary atomic operations
        atomicAdd(&(hist_out[tx]), smem[tx]);
}