#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ConvolutionRowGPU(double *d_Dst, double *d_Src, double *d_Filter, int imageW, int imageH, int filterR) {
    // Use shared memory to reduce global memory access
    extern __shared__ double shared_Src[];

    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int local_col = threadIdx.x;

    // Load data into shared memory
    if (row < imageH && local_col < imageW) {
        shared_Src[local_col] = d_Src[row * imageW + col];
    }
    __syncthreads(); // Ensure all threads have loaded data into shared memory

    double sum = 0;

    // Perform convolution using shared memory
    for (int k = -filterR; k <= filterR; k++) {
        int d = col + k;

        if (d >= 0 && d < imageW) {
            // Access the shared_Src for faster memory access
            sum += shared_Src[local_col + k] * d_Filter[filterR - k];
        }
    }

    // Write the result to global memory
    if (row < imageH && col < imageW) {
        d_Dst[row * imageW + col] = sum;
    }
}