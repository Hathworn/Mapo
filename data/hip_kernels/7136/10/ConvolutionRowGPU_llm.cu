#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ConvolutionRowGPU(float *d_Dst, float *d_Src, float *d_Filter, int filterR) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // Global x index
    int y = blockIdx.y * blockDim.y + threadIdx.y; // Global y index
    int k;

    float sum = 0;
    // Compute convolution only for valid pixels
    for (k = -filterR; k <= filterR; k++) {
        int d = x + k;
        if (d >= 0 && d < gridDim.x * blockDim.x) { // Use grid size to handle border
            sum += d_Src[y * gridDim.x * blockDim.x + d] * d_Filter[filterR - k];
        }
    }
    d_Dst[y * gridDim.x * blockDim.x + x] = sum; // Write to global memory
}