#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogram_equalization(int *lut, unsigned char *img_out, unsigned char *img_in, int *hist_in, int img_size, int nbr_bin) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    
    // Load LUT into shared memory; potentially coalesced memory access
    __shared__ int smem[256];
    if (tx < nbr_bin) {
        smem[tx] = lut[tx];
    }
    __syncthreads();
    
    unsigned int col = tx + blockDim.x * bx;
    unsigned int row = ty + blockDim.y * by;
    int grid_width = gridDim.x * blockDim.x;
    int id = row * grid_width + col;
    
    // Ensure global memory access is within bounds and write directly to img_out
    if (id < img_size) {
        unsigned char pixel_value = img_in[id];
        img_out[id] = smem[pixel_value] > 255 ? 255 : (unsigned char)smem[pixel_value];
    }
}