#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ConvolutionColGPU(float *d_Dst, float *d_Src, float *d_Filter, int filterR) {
    int x = threadIdx.x;
    int y = threadIdx.y;

    float sum = 0;

    // Unroll loop for better performance, assuming filterR is small and known at compile time.
    for (int k = -filterR; k <= filterR; k++) { 
        int d = y + k;
        // Use constant memory for d_Filter if applicable, else assume d_Filter is cached
        if (d >= 0 && d < blockDim.y) {
            sum += d_Src[d * blockDim.x + x] * d_Filter[filterR - k]; 
        }
    }

    // Write once to global memory
    d_Dst[y * blockDim.x + x] = sum;
}