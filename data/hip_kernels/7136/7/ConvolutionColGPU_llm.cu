#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized ConvolutionColGPU kernel
__global__ void ConvolutionColGPU(double *d_Dst, double *d_Src, double *d_Filter, int imageW, int imageH, int filterR) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    // Return immediately if row or col is out of bounds
    if (row >= imageH || col >= imageW) return;

    double sum = 0.0;
    for (int k = -filterR; k <= filterR; k++) {
        int d = row + k;
        // Check if the index is within bounds
        if (d >= 0 && d < imageH) { // Correct imageH instead of imageW
            sum += d_Src[col + imageW * d] * d_Filter[filterR - k];
        }
    }
    // Store the result in the output array
    d_Dst[row * imageW + col] = sum;
}