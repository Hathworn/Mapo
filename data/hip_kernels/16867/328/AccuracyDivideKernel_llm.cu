#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Improved code for better performance and consistency with CUDA guidelines
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (idx == 0) { // Ensure only the first thread executes the division
        *accuracy /= N;
    }
}