#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float local_sum = 0.0f;

    // Efficient parallel reduction to compute the inner products
    for (int i = idx; i < dim; i += blockDim.x) {
        local_sum += dY[i] * Y[i];
    }
    reduction_buffer[idx] = local_sum;
    __syncthreads();

    // Warp-level reduction for better performance in reduction phase
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (idx < offset) {
            reduction_buffer[idx] += reduction_buffer[idx + offset];
        }
        __syncthreads();
    }
    
    // Compute gradient using the reduction result
    float dot_product = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - dot_product);
    }
}
```
