#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CELL_PER_THREAD 1

__global__ void kernel_compute_gen_shared(uint8_t *matrix_src, uint8_t *matrix_dst, uint32_t dim_m1, uint32_t cols) {
    extern __shared__ uint8_t shared[];
    uint32_t cell = blockIdx.x * blockDim.x + threadIdx.x;

    uint32_t x1 = cell & (cols - 1);
    uint32_t x0 = (x1 - 1) & (cols - 1);
    uint32_t x2 = (x1 + 1) & (cols - 1);

    uint32_t y1 = cell - x1;
    uint32_t y0 = (y1 - cols) & dim_m1;
    uint32_t y2 = (y1 + cols) & dim_m1;

    // Load center and neighbor cells into shared memory
    shared[threadIdx.x + 1] = matrix_src[x1 + y0];
    shared[threadIdx.x + 1 + blockDim.x + 2] = matrix_src[cell];
    shared[threadIdx.x + 1 + (blockDim.x + 2) * 2] = matrix_src[x1 + y2];

    if (threadIdx.x == 0) {
        shared[0] = matrix_src[x0 + y0];
        shared[blockDim.x + 2] = matrix_src[x0 + y1];
        shared[(blockDim.x + 2) * 2] = matrix_src[x0 + y2];
    } else if (threadIdx.x == blockDim.x - 1) {
        shared[blockDim.x + 1] = matrix_src[x2 + y0]; // Corrected index calculation
        shared[(blockDim.x + 2) * 2 + blockDim.x + 1] = matrix_src[x2 + y2]; // Corrected index calculation
    }

    __syncthreads();

    uint32_t scell = threadIdx.x + 1 + blockDim.x + 2;
    uint32_t sx1 = scell;
    uint32_t sx0 = (sx1 - 1);
    uint32_t sx2 = (sx1 + 1);

    uint32_t sy1 = scell - sx1;
    uint32_t sy0 = (sy1 - blockDim.x - 2);
    uint32_t sy2 = (sy1 + blockDim.x + 2);

    // Count alive neighboring cells
    uint8_t aliveCells = shared[sx0 + sy0] + shared[sx1 + sy0] + shared[sx2 + sy0] +
                         shared[sx0 + sy1] + shared[sx2 + sy1] +
                         shared[sx0 + sy2] + shared[sx1 + sy2] + shared[sx2 + sy2];

    // Update the destination cell based on the number of alive neighbors
    matrix_dst[y1 + x1] = (aliveCells == 3 || (aliveCells == 2 && shared[scell])) ? 1 : 0;
}