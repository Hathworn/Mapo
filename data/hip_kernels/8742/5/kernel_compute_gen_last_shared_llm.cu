#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CELL_PER_THREAD 1

__global__ void kernel_compute_gen_last_shared(uint8_t *matrix_src, uint8_t *matrix_dst, uint32_t rows, uint32_t cols) {
    __shared__ int shared[3][128 + 2]; // Shared memory for a small tile of the grid

    int ix = ((blockDim.x - 2) * blockIdx.x + threadIdx.x) & (cols - 1);
    int iy = (blockIdx.y + threadIdx.y) & (rows - 1);
    int id = iy * cols + ix;

    int i = threadIdx.y; // Thread index in tile block
    int j = threadIdx.x;

    uint8_t mine = matrix_src[id]; // Load cell into register
    shared[i][j] = mine; // Copy cell to shared memory

    __syncthreads(); // Synchronize to ensure all threads have written to shared memory

    if (i == 1 && j > 0 && j < 129) { // Check if thread processes a central cell
        uint8_t aliveCells =
            shared[i + 1][j] +        // lower
            shared[i - 1][j] +        // upper
            shared[i][j + 1] +        // right
            shared[i][j - 1] +        // left
            shared[i + 1][j + 1] +
            shared[i - 1][j - 1] +    // diagonals
            shared[i - 1][j + 1] +
            shared[i + 1][j - 1];

        matrix_dst[id] = (aliveCells == 3 || (aliveCells == 2 && mine)) ? 1 : 0; // Apply the Game of Life rules
    }
}