#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CELL_PER_THREAD 1

__device__ void gpu_swap(void **a, void **b) {
    void *temp = *a;
    *a = *b;
    *b = temp;
}

__global__ void kernel_compute_gen_singleblock(uint8_t *matrix_src, uint8_t *matrix_dst, uint32_t dim_m1, uint32_t cols, uint32_t iterations, uint32_t cellPerThreads) {
    uint32_t start = threadIdx.x * cellPerThreads;
    uint32_t end = cellPerThreads + start;
    uint32_t x1, x0, x2, y1, y0, y2;
    uint8_t aliveCells;

    for (uint32_t iter = 0; iter < iterations; iter++) {
        // Loop unrolling to increase performance
        for (uint32_t cell = start; cell < end; cell++) {
            x1 = cell & cols-1; // % cols
            x0 = (x1 - 1) & cols-1; // % cols
            x2 = (x1 + 1) & cols-1; // % cols
            y1 = cell - x1;
            
            // Pre-calculate indices to avoid repeated computation
            y0 = (y1 - cols) & dim_m1; // % dim
            y2 = (y1 + cols) & dim_m1; // % dim

            // Optimize memory access by calculating aliveCells all at once
            aliveCells = matrix_src[x0 + y0] + matrix_src[x1 + y0] + matrix_src[x2 + y0] +
                         matrix_src[x0 + y1] + matrix_src[x2 + y1] +
                         matrix_src[x0 + y2] + matrix_src[x1 + y2] + matrix_src[x2 + y2];

            matrix_dst[y1 + x1] = (aliveCells == 3 || (aliveCells == 2 && matrix_src[y1 + x1])) ? 1 : 0;
        }

        gpu_swap((void**)&matrix_dst, (void**)&matrix_src);
        __syncthreads();
    }
}