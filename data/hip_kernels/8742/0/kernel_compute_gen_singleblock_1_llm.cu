#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CELL_PER_THREAD 1

__device__ void gpu_swap(void **a, void **b) {
    void *temp = *a;
    *a = *b;
    *b = temp;
}

__global__ void kernel_compute_gen_singleblock_1(uint8_t *matrix_src, uint8_t *matrix_dst, uint32_t dim_m1, uint32_t cols, uint32_t iterations) {
    // Compute cell index considering block size
    uint32_t cell = blockIdx.x * blockDim.x + threadIdx.x;

    // Precompute column and row indices
    uint32_t x1 = cell & (cols - 1);
    uint32_t x0 = (x1 - 1) & (cols - 1);
    uint32_t x2 = (x1 + 1) & (cols - 1);

    uint32_t y1 = cell - x1;
    uint32_t y0 = (y1 - cols) & dim_m1;
    uint32_t y2 = (y1 + cols) & dim_m1;

    for (uint32_t iter = 0; iter < iterations; iter++) {
        // Calculate alive cells using shared memory
        uint8_t aliveCells = matrix_src[x0 + y0] + matrix_src[x1 + y0] + matrix_src[x2 + y0] +
                             matrix_src[x0 + y1] + matrix_src[x2 + y1] +
                             matrix_src[x0 + y2] + matrix_src[x1 + y2] + matrix_src[x2 + y2];

        // Update the matrix
        matrix_dst[y1 + x1] = (aliveCells == 3 || (aliveCells == 2 && matrix_src[y1 + x1])) ? 1 : 0;
        
        // Only swap pointers if needed outside the loop
        if(threadIdx.x == 0) {
            gpu_swap((void**)&matrix_dst, (void**)&matrix_src);
        }
        __syncthreads();
    }
}