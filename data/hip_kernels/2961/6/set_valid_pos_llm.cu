#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_valid_pos(int32_t* pos_buff, int32_t* count_buff, const int32_t entry_count) {
    const int32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    const int32_t stride = blockDim.x * gridDim.x;
    
    // Use shared memory to improve memory access efficiency
    extern __shared__ int32_t shared_count_buff[];
    
    for (int32_t i = idx; i < entry_count; i += stride) {
        shared_count_buff[threadIdx.x] = count_buff[i];  // Load into shared memory
        __syncthreads();  // Ensure all loads are completed

        if (VALID_POS_FLAG == pos_buff[i]) {
            // Efficient computation using shared memory
            pos_buff[i] = !i ? 0 : shared_count_buff[threadIdx.x - 1];
        }
        
        __syncthreads();  // Ensure all writes are completed
    }
}