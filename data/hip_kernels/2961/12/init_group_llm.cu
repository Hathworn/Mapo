#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_group(int8_t* groups, const size_t group_count, const size_t col_count, const size_t* col_widths, const size_t* init_vals) {
    // Calculate the flattened thread index
    const auto thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index does not exceed group count
    if (thread_index >= group_count) {
        return;
    }
    
    // Iterate over columns
    for (size_t i = 0, offset = 0; i < col_count; offset += col_widths[i++] * group_count) {
        int8_t* group_ptr = groups + offset;
        switch (col_widths[i]) {
            case 4:
                // Efficiently handle 4-byte width using reinterpret cast
                *reinterpret_cast<uint32_t*>(group_ptr + thread_index * 4) =
                    *reinterpret_cast<const uint32_t*>(init_vals + i);
                break;
            case 8:
                // Efficiently handle 8-byte width with correct indexing
                reinterpret_cast<size_t*>(group_ptr)[thread_index] = init_vals[i];
                break;
            default:
                break;
        }
    }
}