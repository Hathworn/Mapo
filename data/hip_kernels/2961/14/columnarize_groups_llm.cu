#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void columnarize_groups(int8_t* columnar_buffer, const int8_t* rowwise_buffer, const size_t row_count, const size_t col_count, const size_t* col_widths, const size_t row_size) {
    const auto thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_index >= row_count) {
        return;
    }

    auto read_ptr = rowwise_buffer + thread_index * row_size;
    auto col_base = columnar_buffer;

    for (size_t i = 0; i < col_count; ++i) {
        switch (col_widths[i]) {
            case 8: {
                // Use int64_t pointer for 8-byte wide data
                reinterpret_cast<int64_t*>(col_base)[thread_index] = *reinterpret_cast<const int64_t*>(read_ptr);
                break;
            }
            case 4: {
                // Use int32_t pointer for 4-byte wide data
                reinterpret_cast<int32_t*>(col_base)[thread_index] = *reinterpret_cast<const int32_t*>(read_ptr);
                break;
            }
            default:; // Handle other cases if needed
        }
        col_base += col_widths[i] * row_count;
        read_ptr += col_widths[i];
    }
}