#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Reduced indexing computation and used register memory for `entry_count`
__global__ void set_valid_pos_flag(int32_t* pos_buff, const int32_t* count_buff, const int32_t entry_count) {
    int32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    int32_t stride = blockDim.x * gridDim.x;
    while (idx < entry_count) {
        if (count_buff[idx]) {
            pos_buff[idx] = VALID_POS_FLAG;
        }
        idx += stride; // Increment using stride once, removing repetitive calculations
    }
}