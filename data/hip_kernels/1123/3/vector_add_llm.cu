#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vector_add(float *out, float *a, float *b, int n) {
// Improve parallelism by using blockIdx and gridDim for better workload distribution
int index = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;

for(int i = index; i < n; i += stride){
    out[i] = a[i] + b[i];
}
}