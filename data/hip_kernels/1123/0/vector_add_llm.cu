#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
    // Calculate the global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform vector addition if idx is within range
    if (idx < n) {
        out[idx] = a[idx] + b[idx];
    }
}