#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vector_add(float *out, float *a, float *b, int n) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Each thread processes one element
    if (idx < n) {
        out[idx] = a[idx] + b[idx];
    }
}