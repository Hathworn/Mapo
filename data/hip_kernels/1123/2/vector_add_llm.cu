#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vector_add(float *out, float *a, float *b, int n) {
    // Use shared memory for efficient memory access
    __shared__ float shared_a[1024]; // Example size, adjust based on block size
    __shared__ float shared_b[1024]; // Example size, adjust based on block size

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    // Load array elements into shared memory
    if (tid < n) {
        shared_a[local_tid] = a[tid];
        shared_b[local_tid] = b[tid];
        __syncthreads();  // Ensure complete loading before proceeding

        // Perform addition using shared memory
        out[tid] = shared_a[local_tid] + shared_b[local_tid];
    }
}