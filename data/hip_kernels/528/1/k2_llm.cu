#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Update frontier
__global__ void k2(const int N, bool* visited, int* frontier, bool* new_frontier) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID
    __shared__ int count_shared; // Shared variable to count elements in the frontier

    if (threadIdx.x == 0) {
        count_shared = 0; // Initialize shared count in the first thread of each block
    }
    __syncthreads();

    int local_count = 0; // Local count for this thread
    if (thread_id < N && new_frontier[thread_id]) {
        new_frontier[thread_id] = false; // Clear new_frontier entry
        int index = atomicAdd(&count_shared, 1); // Atomically get the index for this thread
        frontier[index + 1] = thread_id; // Assign frontier index, offset by 1
        visited[thread_id] = true; // Mark as visited
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        atomicAdd(&frontier[0], count_shared); // Update global count of frontier elements
    }
}