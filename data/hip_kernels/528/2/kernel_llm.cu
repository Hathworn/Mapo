#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(bool* adj_mat, const int N, bool* visited, int* frontier, bool* new_frontier, bool* par_mat, int* cap_mat, int* cap_max_mat) {
    int row_idx = frontier[blockIdx.x+1];
    long offset = N * row_idx;

    int col_idx = threadIdx.x;

    // Cache adj_mat and visited values to reduce global memory accesses
    bool adj_val = adj_mat[offset + col_idx];
    bool not_visited = !visited[col_idx];

    // Use a single conditional check to improve data locality and avoid code duplication
    if(adj_val && not_visited) {
        long offset2 = N * col_idx;
        bool adjacency_check = cap_mat[offset + col_idx] < cap_max_mat[offset + col_idx];
        bool capacity_check = cap_mat[offset2 + row_idx] > 0;

        if(adjacency_check || capacity_check) {
            new_frontier[col_idx] = true;
            par_mat[offset2 + row_idx] = true;
        }
    }
}