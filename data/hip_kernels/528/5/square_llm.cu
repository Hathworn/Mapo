#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square(float * d_out, float * d_in)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Compute global index for threads
    float f = d_in[idx];
    d_out[idx] = f * f; // Square the input value and store in output
}