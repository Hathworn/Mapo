#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void k2(const int N, bool* visited, int* frontier, bool* new_frontier, bool* augFound) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique thread ID
    __shared__ int local_count; // Shared memory for count
    if (threadIdx.x == 0) local_count = 0; // Initialize shared variable in block

    __syncthreads();

    if (tid < N && new_frontier[tid]) {
        int index = atomicAdd(&local_count, 1); // Atomic add to avoid race conditions
        new_frontier[tid] = false;
        frontier[index + 1] = tid;
        visited[tid] = true;
    }
    
    __syncthreads();

    if (threadIdx.x == 0) frontier[0] = local_count; // Only one thread writes to global memory

    __syncthreads();

    if (tid < frontier[0] && frontier[tid + 1] == (N - 1)) {
        augFound[0] = true; // Check if sink has been reached
    }
}