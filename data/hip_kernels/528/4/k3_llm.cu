#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void k3(const int N, int* augPath, bool* visited, int* frontier, bool* new_frontier, bool* par_mat, int* cap_mat, bool* adj_mat, int* cap_max_mat, int* maxflow, bool* augFound) {
    augFound[0] = false;

    // Optimized path finding with while loop
    augPath[0] = N - 1;
    int i = 1, vertex = N - 1;
    while (vertex != 0) {
        int found = 0;
        for (int j = 0; j < N && !found; j++) {
            if (par_mat[vertex * N + j]) {
                vertex = j;
                augPath[i] = vertex;
                i++;
                found = 1;
            }
        }
    }

    // Computing bottleneck more effectively
    int bottleneck = INT_MAX;
    for (int i = 0; augPath[i] != 0 && i < N - 1; i++) {
        int k = augPath[i];
        int j = augPath[i + 1];
        int freeCap = adj_mat[j * N + k] ? (cap_max_mat[j * N + k] - cap_mat[j * N + k]) : cap_mat[k * N + j];
        bottleneck = min(bottleneck, freeCap);
    }
    maxflow[0] += bottleneck;

    // Update capacities based on the bottleneck
    for (int i = 0; augPath[i] != 0 && i < N - 1; i++) {
        int k = augPath[i];
        int j = augPath[i + 1];
        if (adj_mat[j * N + k]) {
            cap_mat[j * N + k] += bottleneck;
        } else {
            cap_mat[k * N + j] -= bottleneck;
        }
    }

    // Initialize matrices and vectors
    for (int i = 0; i < N * N; i++) par_mat[i] = false;
    for (int i = 0; i < N; i++) visited[i] = false;
    for (int i = 0; i < N; i++) new_frontier[i] = false;

    visited[0] = true;
    frontier[0] = 1;
    frontier[1] = 0;
}