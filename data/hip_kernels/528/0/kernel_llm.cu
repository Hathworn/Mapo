#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

//Check for edges valid to be part of augmented path

//Update frontier

__global__ void kernel(bool* adj_mat, const int N, bool* visited, int* frontier, bool* new_frontier, bool* par_mat, int* cap_mat, int* cap_max_mat) {
    // Obtain row index from frontier using grid stride loop
    int block_offset = blockDim.x * blockIdx.x;
    for (int row_shift = block_offset + threadIdx.x; row_shift < blockDim.x * (blockIdx.x + 1); row_shift += blockDim.x) {
        int row_idx = frontier[row_shift + 1];
        long row_offset = N * row_idx;

        // Use shared memory for col_idx computation
        extern __shared__ int shared_row_idxs[];
        shared_row_idxs[threadIdx.x] = row_idx;
        __syncthreads();

        // Process columns
        for (int col_idx = 0; col_idx < N; ++col_idx) {
            long offset2 = N * col_idx;
            if (adj_mat[row_offset + col_idx] && (cap_mat[row_offset + col_idx] < cap_max_mat[row_offset + col_idx]) && !visited[col_idx]) {
                new_frontier[col_idx] = true;
                par_mat[offset2 + shared_row_idxs[threadIdx.x]] = true;
            }

            if (adj_mat[offset2 + row_idx] && (cap_mat[offset2 + row_idx] > 0) && !visited[col_idx]) {
                new_frontier[col_idx] = true;
                par_mat[offset2 + shared_row_idxs[threadIdx.x]] = true;
            }
        }
    }
}