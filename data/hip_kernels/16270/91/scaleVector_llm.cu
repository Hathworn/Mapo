#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Calculate the global thread index and stride
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    // Use striding for better load balancing and coalescing
    for (int i = pos; i < len; i += stride)
    {
        d_res[i] = d_src[i] * scale;
    }
}