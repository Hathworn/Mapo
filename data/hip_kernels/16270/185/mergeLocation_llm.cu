#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Utilize shared memory for coalesced access and improved performance
    extern __shared__ short2 sharedLoc[];

    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        sharedLoc[threadIdx.x] = loc_[ptidx]; // Load into shared memory

        __syncthreads(); // Ensure all threads have loaded their data

        const short2 loc = sharedLoc[threadIdx.x]; // Access from shared memory

        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}