#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16  // Define block size for better shared memory utilization

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Compute corner of target pixel
    float cx = u_ * time_scale + (float)j + 1.0f;
    float cy = v_ * time_scale + (float)i + 1.0f;

    // Convert to integer coordinates
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Read source value
    float value = src[image_row_offset + j];

    // Check bounds and atomically add to destination
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        atomicAdd(dst + ty * image_stride + tx, value); // Use atomicAdd directly
    }
}