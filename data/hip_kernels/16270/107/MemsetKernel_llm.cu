#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

// Optimized kernel
__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < h && j < w) // Avoids out-of-bounds threads
    {
        const int pos = i * w + j;
        image[pos] = value; // Set value directly
    }
}