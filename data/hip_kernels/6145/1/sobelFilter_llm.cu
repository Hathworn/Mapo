#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#define WIDTH 256
#define HEIGHT 256

#define BLOCK_W 16
#define BLOCK_H 16

float image[HEIGHT][WIDTH];
float final[HEIGHT][WIDTH];

void load_image();
void call_kernel();
void save_image();

#define MAXLINE 128

float total, sobel;
hipEvent_t start_total, stop_total;
hipEvent_t start_sobel, stop_sobel;

__global__ void sobelFilter(float *input, float *output, int width, int height) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (row < height && col < width && row > 0 && col > 0) {
        // Use numcols directly inside the kernel for simplicity
        const int numcols = WIDTH;

        // Precompute indices to avoid redundant calculations
        int idx = row * numcols + col;
        int idx_up = (row - 1) * numcols + col;
        int idx_down = (row + 1) * numcols + col;
        int idx_left = row * numcols + (col - 1);
        int idx_right = row * numcols + (col + 1);

        // Preload shared data
        float x0 = input[idx_up - 1];  // leftup
        float x1 = input[idx_up];      // up
        float x2 = input[idx_up + 1];  // rightup
        float x3 = input[idx_left];    // left
        float x5 = input[idx_right];   // right
        float x6 = input[idx_down - 1]; // leftdown
        float x7 = input[idx_down];    // down
        float x8 = input[idx_down + 1]; // rightdown

        // Compute gradients
        float gradient_h = (x0 * -1) + (x2 * 1) + (x3 * -2) + (x5 * 2) + (x6 * -1) + (x8 * 1);
        float gradient_v = (x0 * -1) + (x1 * -2) + (x3 * -1) + (x6 * 1) + (x7 * 2) + (x8 * 1);

        // Use local variable for threshold comparison
        const float threshold = 30;
        float gradient = sqrt((gradient_h * gradient_h) + (gradient_v * gradient_v));

        // Threshold logic using conditional assignment
        output[idx] = (gradient >= threshold) ? 255.0f : 0.0f;
    }
}