#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blend_kernel(float *outSrc, const float *inSrc) {
    // Calculate global thread ID directly
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // Use ternary operators for boundary checks
    int left = (x > 0) ? offset - 1 : offset;
    int right = (x < DIM - 1) ? offset + 1 : offset;
    int top = (y > 0) ? offset - DIM : offset;
    int bottom = (y < DIM - 1) ? offset + DIM : offset;

    // Compute output using optimized access
    outSrc[offset] = inSrc[offset] + SPEED * (inSrc[top] + inSrc[bottom] + inSrc[left] + inSrc[right] - inSrc[offset] * 4);
}