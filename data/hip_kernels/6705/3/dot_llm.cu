#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use templated shared memory and optimize block synchronization
__global__ void dot(float* a, float* b, float* c) {
    extern __shared__ float cache[];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    
    float temp = 0.0f;

    // Efficient iteration with stride
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        temp += a[i] * b[i];
    }

    // Store partial sum in shared memory
    cache[cacheIndex] = temp;
    __syncthreads();

    // Reduce within block using binary tree reduction
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    // Write block result to global memory
    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}
```
