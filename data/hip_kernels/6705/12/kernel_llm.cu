#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(unsigned char *ptr, int ticks) {
    // Efficiently calculate pixel positions
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // Calculate distance from the center
    float fx = x - DIM / 2;
    float fy = y - DIM / 2;
    float d = rsqrtf(fx * fx + fy * fy);  // Use reciprocal square root for efficiency

    // Calculate grey value
    float value = cosf(d * 10.0f - ticks / 7.0f) / (1.0f / d + 1.0f);
    unsigned char grey = (unsigned char)(128.0f + 127.0f * value);

    // Assign grey value to each channel, set alpha to fully opaque
    int pixelIndex = offset * 4;
    ptr[pixelIndex + 0] = grey;
    ptr[pixelIndex + 1] = grey;
    ptr[pixelIndex + 2] = grey;
    ptr[pixelIndex + 3] = 255;
}