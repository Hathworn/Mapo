#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_const_kernel(float *iptr, const float *cptr) {
    // Calculate offset in a 1D fashion to maximize memory coalescing
    int offset = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * (blockIdx.y + gridDim.y * blockIdx.x));

    // Conditional copy to iptr only if cptr is non-zero
    if (cptr[offset] != 0) {
        iptr[offset] = cptr[offset];
    }
}