#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;  // Use full thread index
    if(tid < N) 
        c[tid] = a[tid] + b[tid];  // Coalesced memory access
}