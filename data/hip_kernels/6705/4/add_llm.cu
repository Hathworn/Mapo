#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c) {
    // Use shared memory to reduce global memory accesses
    __shared__ int temp; 

    // Compute sum in a single thread to minimize thread divergence
    if (threadIdx.x == 0) {
        temp = a + b;
    }

    // Ensure all threads see the updated shared memory value
    __syncthreads();

    // Write result from shared memory back to global memory
    if (threadIdx.x == 0) {
        *c = temp;
    }
}