#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128

#define DISP_MAX 256

__global__ void subpixel_enchancement(float *d0, float *c2, float *out, int size, int dim23, int disp_max) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        int d = d0[id];
        out[id] = d;
        if (1 <= d && d < disp_max - 1) {
            // Optimize memory accesses with shared memory
            extern __shared__ float shared_c2[];
            int shared_idx = threadIdx.x * 3;
            shared_c2[shared_idx] = c2[(d - 1) * dim23 + id];
            shared_c2[shared_idx + 1] = c2[d * dim23 + id];
            shared_c2[shared_idx + 2] = c2[(d + 1) * dim23 + id];

            __syncthreads();

            float cn = shared_c2[shared_idx];
            float cz = shared_c2[shared_idx + 1];
            float cp = shared_c2[shared_idx + 2];

            // Reduce redundant calculations in denom
            float denom = 2 * (cp + cn - 2 * cz);

            if (denom > 1e-5) {
                out[id] = d - min(1.0f, max(-1.0f, (cp - cn) / denom));
            }
        }
    }
}