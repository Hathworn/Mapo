#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void remove_nonvisible(float *y, int size, int size3)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        // Calculate x more efficiently with integer division and remainder
        int x = id % size3;
        float value = y[id]; // Cache the value to avoid repeated memory access
        if (value >= x) {
            y[id] = 0;
        }
    }
}