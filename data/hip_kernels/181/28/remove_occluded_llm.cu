#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void remove_occluded(float *y, int size, int size3)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        int x = id % size3;
        // Use shared memory to optimize repeated access
        __shared__ float shared_y[TB];
        shared_y[threadIdx.x] = y[id];

        __syncthreads();

        for (int i = 1; x + i < size3; i++) {
            // Reduce global memory access by using shared memory 
            if (i - shared_y[threadIdx.x + i] < -shared_y[threadIdx.x]) {
                y[id] = 0;
                break;
            }
        }
    }
}