#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_SpatialLogSoftMax_updateOutput_kernel(float *output, float *input, int feature_size, int spatial_size, int data_size, float constant) {
    // Calculate global index
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = (global_idx / spatial_size) * feature_size + global_idx % spatial_size;

    if (global_idx < data_size) {
        int next_idx = idx + feature_size;
        float logsum = 0.0f;
        float max_val = -2e38f;

        // Unroll loops to improve parallel execution efficiency
#pragma unroll
        for (int i = idx; i < next_idx; i += spatial_size) {
            max_val = fmaxf(max_val, input[i]);
        }

#pragma unroll
        for (int i = idx; i < next_idx; i += spatial_size) {
            if (!isnan(input[i])) {
                logsum += __expf(input[i] - max_val);
            }
        }
        logsum += constant;
        logsum = __logf(logsum) + max_val;

        // Assign computed values to output
#pragma unroll
        for (int i = idx; i < next_idx; i += spatial_size) {
            output[i] = input[i] - logsum;
        }
    }
}