#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_SpatialLogSoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int feature_size, int spatial_size, int data_size)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int base_idx = (idx / spatial_size) * feature_size + idx % spatial_size;

    // Boundary check
    if (base_idx < data_size) {
        int next_idx = base_idx + feature_size;
        float gradSum = 0.0f;

        // Use loop unrolling for optimization
        #pragma unroll
        for (int i = base_idx; i < next_idx; i += spatial_size) {
            gradSum += gradOutput[i];
        }

        // Use loop unrolling for optimization
        #pragma unroll
        for (int i = base_idx; i < next_idx; i += spatial_size) {
            gradInput[i] = gradOutput[i] - __expf(output[i]) * gradSum;
        }
    }
}