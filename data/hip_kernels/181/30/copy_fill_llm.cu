#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void copy_fill(float *in, float *out, int size, int in_size2, int in_size3, int out_size2, int out_size3)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        int out_x = id % out_size3;
        int out_y = id / out_size3;

        // Precompute the offset values to reduce redundant calculations
        int offset_x = (out_size3 - in_size3) / 2;
        int offset_y = (out_size2 - in_size2) / 2;

        int in_x = out_x - offset_x;
        int in_y = out_y - offset_y;

        // Use clamp to keep values in range for better readability
        int x = max(0, min(in_size3 - 1, in_x));
        int y = max(0, min(in_size2 - 1, in_y));

        out[id] = in[y * in_size3 + x];
    }
}