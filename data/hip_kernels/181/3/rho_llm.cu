#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void rho(float *x, int size, float lambda)
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop to process larger data sets
    int stride = blockDim.x * gridDim.x;
    for (int i = id; i < size; i += stride) {
        x[i] = 1 - expf(-x[i] / lambda); // Use expf for single precision
    }
}