#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void remove_white(float *x, float *y, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride to process more elements

    for (int i = id; i < size; i += stride) { // Use loop to handle larger problem sizes
        if (x[i] == 255.0f) { // Ensure correct float comparison
            y[i] = 0.0f; // Avoid partial write, use float value
        }
    }
}