#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128

#define DISP_MAX 256

__global__ void Normalize_get_norm_(float *input, float *norm, int size1, int size23, int size023)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for reduction to minimize global memory accesses
    extern __shared__ float sharedSum[];

    if (id < size023) {
        int dim23 = id % size23;
        int dim0 = id / size23;

        float sum = 0.0f;
        for (int dim1 = threadIdx.y; dim1 < size1; dim1 += blockDim.y) {
            float x = input[(dim0 * size1 + dim1) * size23 + dim23];
            sum += x * x;
        }

        sharedSum[threadIdx.x] = sum;
        __syncthreads();

        // Intra-block reduction
        for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
            if (threadIdx.x < offset) {
                sharedSum[threadIdx.x] += sharedSum[threadIdx.x + offset];
            }
            __syncthreads();
        }

        // Write the final result to global memory
        if (threadIdx.x == 0) {
            norm[dim0 * size23 + dim23] = sharedSum[0] + 1e-5f;
        }
    }
}