#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    int batch_id = blockIdx.x; // Assign unique ID for each batch
    int point_id = blockIdx.y * blockDim.x + threadIdx.x; // Assign unique ID for each point in xyz2 using thread index

    if (batch_id < b && point_id < m) {
        int cnt = 0;
        float x2 = xyz2[point_id * 3 + 0];
        float y2 = xyz2[point_id * 3 + 1];
        float z2 = xyz2[point_id * 3 + 2];
        
        for (int k = 0; k < n; ++k) {
            if (cnt == nsample)
                break; // only pick the FIRST nsample points in the ball

            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            float d_sq = (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1);
            
            if (d_sq < radius * radius) {
                if (cnt == 0) { // set ALL indices to k, if less points in ball than nsample, use valid indices
                    for (int l = 0; l < nsample; ++l)
                        idx[point_id * nsample + l] = k;
                }
                idx[point_id * nsample + cnt] = k;
                cnt += 1;
            }
        }
    }
}