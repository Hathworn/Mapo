#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, float *dist, int *idx, float *val) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    idx += m * k * batch_index;
    val += m * k * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int j = index; j < m; j += stride) {
        float *p_dist = dist + j * n;
        // Use shared memory for current element set
        __shared__ float shared_dist[1024];
        int np = n;
        if (np > 1024) np = 1024; // Ensure shared memory size is not exceeded
        for (int i = threadIdx.x; i < np; i += blockDim.x) {
            shared_dist[i] = p_dist[i];
        }
        __syncthreads();

        // Selection sort for the first k elements using shared memory
        for (int s = 0; s < k; ++s) {
            int min = s;
            for (int t = s + 1; t < n; ++t) {
                if (shared_dist[t] < shared_dist[min]) {
                    min = t;
                }
            }
            idx[j * n + s] = min;
            val[j * n + s] = shared_dist[min];
            float tmp = shared_dist[min];
            shared_dist[min] = shared_dist[s];
            shared_dist[s] = tmp;
        }

        // Write back modified portion to global memory
        for (int i = threadIdx.x; i < np; i += blockDim.x) {
            p_dist[i] = shared_dist[i];
        }
    }
}