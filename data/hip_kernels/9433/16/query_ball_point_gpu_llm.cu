#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    int batch_index = blockIdx.x;
    int index = threadIdx.x;
    int stride = blockDim.x;
    xyz1 += n * 3 * batch_index;
    xyz2 += m * 3 * batch_index;
    idx += m * nsample * batch_index;
    
    // Use shared memory for frequently accessed data
    extern __shared__ float shared_xyz2[];

    for (int j = index; j < m; j += stride) {
        shared_xyz2[j * 3 + 0] = xyz2[j * 3 + 0];
        shared_xyz2[j * 3 + 1] = xyz2[j * 3 + 1];
        shared_xyz2[j * 3 + 2] = xyz2[j * 3 + 2];
    }
    __syncthreads();

    for (int j = index; j < m; j += stride) {
        int cnt = 0;
        float x2 = shared_xyz2[j * 3 + 0];
        float y2 = shared_xyz2[j * 3 + 1];
        float z2 = shared_xyz2[j * 3 + 2];
        for (int k = 0; k < n; ++k) {
            if (cnt == nsample)
                break; // Early termination if nsample is reached
            
            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            float dist_squared = (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1);
            
            if (dist_squared < radius * radius) { // Compare squared distances to avoid sqrt
                if (cnt == 0) {
                    for (int l = 0; l < nsample; ++l)
                        idx[j * nsample + l] = k;
                }
                idx[j * nsample + cnt] = k;
                cnt += 1;
            }
        }
    }
}