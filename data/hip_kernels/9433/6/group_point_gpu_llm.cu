#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    // Calculate batch, index, and stride
    int batch_index = blockIdx.x;
    int index = threadIdx.x;
    int stride = blockDim.x;

    // Adjust pointers for the current batch
    points += n * c * batch_index;
    idx += m * nsample * batch_index;
    out += m * nsample * c * batch_index;

    // Use threads to handle multiple samples per group point
    for (int j = index; j < m; j += stride) {
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];  // Sample index
            float *out_ptr = &out[j * nsample * c + k * c];  // Pointer to output location
            const float *point_ptr = &points[ii * c];  // Pointer to point location

            // Copy data using pointer arithmetic for efficient memory operations
            for (int l = 0; l < c; ++l) {
                out_ptr[l] = point_ptr[l];
            }
        }
    }
}