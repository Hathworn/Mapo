#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    outi += m * n * batch_index;
    out += m * n * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Directly copy data using parallel threads
    for (int j = index; j < m; j += stride) {
        for (int s = 0; s < n; ++s) {
            out[j * n + s] = dist[j * n + s];
            outi[j * n + s] = s;
        }
    }

    __syncthreads(); // Ensure all writes to out and outi are complete

    for (int j = index; j < m; j += stride) {
        float *p_dist = out + j * n;
        // Perform selection sort for the first k elements
        for (int s = 0; s < k; ++s) {
            int min = s;
            for (int t = s + 1; t < n; ++t) {
                if (p_dist[t] < p_dist[min]) {
                    min = t;
                }
            }
            // Swap minimum element with s-th element (avoiding unnecessary swaps)
            if (min != s) {
                float tmp = p_dist[min];
                p_dist[min] = p_dist[s];
                p_dist[s] = tmp;
                int tmpi = outi[j * n + min];
                outi[j * n + min] = outi[j * n + s];
                outi[j * n + s] = tmpi;
            }
        }
    }
}