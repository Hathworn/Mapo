#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    outi += m * n * batch_index;
    out += m * n * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Copy from dist to dist_out
    for (int j = index; j < m; j += stride) {
        for (int s = 0; s < n; ++s) {
            out[j * n + s] = dist[j * n + s];
            outi[j * n + s] = s;
        }
    }

    __shared__ float shared_out[1024]; // Use shared memory for faster access

    for (int j = index; j < m; j += stride) {
        for (int s = 0; s < n; ++s) {
            int global_idx = j * n + s;
            if (s < 1024) shared_out[s] = out[global_idx]; // Load data to shared memory
        }
        __syncthreads();

        float* p_dist = j % (blockDim.x / stride) == 0 ? shared_out : out + j * n; // Decide on using shared/global depending on size
        // Selection sort for the first k elements
        for (int s = 0; s < k; ++s) {
            int min = s;
            // Find the min
            for (int t = s + 1; t < n; ++t) {
                if (p_dist[t] < p_dist[min]) {
                    min = t;
                }
            }
            // Swap min-th and i-th element
            if (min != s) {
                float tmp = p_dist[min];
                p_dist[min] = p_dist[s];
                p_dist[s] = tmp;
                int tmpi = outi[j * n + min];
                outi[j * n + min] = outi[j * n + s];
                outi[j * n + s] = tmpi;
            }
        }
    }
}
```
