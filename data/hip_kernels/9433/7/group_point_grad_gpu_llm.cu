#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_index = blockIdx.x;
    idx += m * nsample * batch_index;
    grad_out += m * nsample * c * batch_index;
    grad_points += n * c * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Use shared memory to minimize global memory access
    extern __shared__ float shared_grad_out[];
    for (int j = index; j < m * nsample * c; j += stride) {
        shared_grad_out[j] = grad_out[j];
    }
    __syncthreads();

    for (int j = index; j < m; j += stride) {
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            for (int l = 0; l < c; ++l) {
                // Use shared memory for grad_out
                atomicAdd(&grad_points[ii * c + l], shared_grad_out[j * nsample * c + k * c + l]);
            }
        }
    }
}