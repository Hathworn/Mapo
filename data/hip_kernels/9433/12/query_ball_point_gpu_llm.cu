#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    int batch_index = blockIdx.x;
    xyz1 += n * 3 * batch_index;
    xyz2 += m * 3 * batch_index;
    idx += m * nsample * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Use shared memory to store selected indices temporarily
    extern __shared__ int shared_idx[];
    
    for (int j = index; j < m; j += stride) {
        int cnt = 0;
        float x2 = xyz2[j * 3 + 0];
        float y2 = xyz2[j * 3 + 1];
        float z2 = xyz2[j * 3 + 2];

        for (int k = 0; k < n; ++k) {
            if (cnt == nsample)
                break; // only pick the FIRST nsample points in the ball
            
            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            float d = max(sqrtf((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1)), 1e-20f);
            
            if (d < radius) {
                if (cnt == 0) { // set ALL indices to k, handle cases with less points than nsample
                    for (int l = 0; l < nsample; ++l)
                        shared_idx[j * nsample + l] = k;
                }
                shared_idx[j * nsample + cnt] = k;
                cnt += 1;
            }
        }
        
        // Copy results from shared memory to global memory
        for (int l = 0; l < nsample; ++l) {
            idx[j * nsample + l] = shared_idx[j * nsample + l];
        }
    }
}