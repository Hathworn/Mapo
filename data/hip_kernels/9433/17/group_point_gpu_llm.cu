#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= b) return; // Ensure the index does not exceed bounds

    points += n * c * index;
    idx += m * nsample * index;
    out += m * nsample * c * index;

    // Loop unrolling for better performance
    for (int j = 0; j < m; ++j) {
        int j_offset_out = j * nsample * c;
        int j_offset_idx = j * nsample;
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j_offset_idx + k];
            int k_offset_out = k * c;
            for (int l = 0; l < c; ++l) {
                out[j_offset_out + k_offset_out + l] = points[ii * c + l];
            }
        }
    }
}