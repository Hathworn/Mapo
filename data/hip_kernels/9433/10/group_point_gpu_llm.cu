#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_idx = blockIdx.x; // Utilize thread blocks for batches
    int sample_idx = threadIdx.x; // Utilize threads within block for samples

    if (batch_idx < b && sample_idx < nsample) {
        const float* batch_points = points + batch_idx * n * c;
        const int* batch_idx_ptr = idx + batch_idx * m * nsample;
        float* batch_out = out + batch_idx * m * nsample * c;

        for (int j = 0; j < m; ++j) {
            int ii = batch_idx_ptr[j * nsample + sample_idx];
            for (int l = 0; l < c; ++l) {
                batch_out[j * nsample * c + sample_idx * c + l] = batch_points[ii * c + l];
            }
        }
    }
}