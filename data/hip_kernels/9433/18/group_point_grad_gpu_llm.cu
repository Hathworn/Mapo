#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batchIdx = blockIdx.x;
    int threadIdx = threadIdx.x;
    
    // Calculate global index for each thread
    int globalThreadIdx = batchIdx * blockDim.x + threadIdx;
    
    // Adjust pointers for current batch
    grad_out += m * nsample * c * globalThreadIdx;
    idx += m * nsample * globalThreadIdx;
    grad_points += n * c * globalThreadIdx;

    // Iterate over each 'm' and 'nsample' for current batch
    for (int j = 0; j < m; ++j) {
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            for (int l = 0; l < c; ++l) {
                // Efficient memory access
                atomicAdd(&grad_points[ii * c + l], grad_out[j * nsample * c + k * c + l]);
            }
        }
    }
}