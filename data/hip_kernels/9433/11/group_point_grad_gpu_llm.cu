#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int j = blockIdx.x;
    int k = threadIdx.x;

    if (j < m && k < nsample) {
        for (int i = 0; i < b; ++i) {
            int ii = idx[j * nsample + k];
            for (int l = 0; l < c; ++l) {
                atomicAdd(&grad_points[ii * c + l], grad_out[j * nsample * c + k * c + l]);
            }
            idx += m * nsample;
            grad_out += m * nsample * c;
            grad_points += n * c;
        }
    }
}