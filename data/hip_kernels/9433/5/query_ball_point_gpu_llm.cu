#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx, int *pts_cnt) {
    int batch_index = blockIdx.x;
    xyz1 += n * 3 * batch_index;
    xyz2 += m * 3 * batch_index;
    idx += m * nsample * batch_index;
    pts_cnt += m * batch_index; // counting how many unique points selected in local region

    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int j = index; j < m; j += stride) {
        int cnt = 0;
        float x2 = xyz2[j * 3 + 0];
        float y2 = xyz2[j * 3 + 1];
        float z2 = xyz2[j * 3 + 2];

        // Pre-fetch xyz2 coordinates outside the loop to improve memory access
        for (int k = 0; k < n && cnt < nsample; ++k) {
            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            float dx = x2 - x1;
            float dy = y2 - y1;
            float dz = z2 - z1;
            float d = sqrtf(dx * dx + dy * dy + dz * dz);

            if (d < radius) {
                if (cnt == 0) {
                    // Improved efficiency by removing redundant if-check for cnt == 0 inside the inner loop
                    for (int l = 0; l < nsample; ++l)
                        idx[j * nsample + l] = k;
                }
                idx[j * nsample + cnt] = k;
                cnt += 1;
            }
        }
        pts_cnt[j] = cnt;
    }
}