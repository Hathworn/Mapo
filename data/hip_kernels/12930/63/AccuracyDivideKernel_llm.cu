#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
  // Use atomic operation for division to ensure thread safety
  atomicExch(accuracy, *accuracy / N);
}