#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    extern __shared__ float reduction_buffer[]; // Use dynamic shared memory
    const int idx = threadIdx.x;
    const int offset = blockIdx.x * dim;
    Y += offset;
    dY += offset;
    dX += offset;
    
    // Optimized reduction using grid-stride loop
    float tmp = 0;
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Reduction using warp shuffle for better efficiency
    if (idx < warpSize) {
        for (int i = warpSize; i < blockDim.x; i += warpSize) {
            if (idx + i < blockDim.x) tmp += reduction_buffer[idx + i];
        }
        for (int offset = warpSize/2; offset > 0; offset /= 2) {
            tmp += __shfl_down(tmp, offset);
        }
    }

    if (idx == 0) reduction_buffer[0] = tmp;
    __syncthreads();
    
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}