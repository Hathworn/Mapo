#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define blockx 16
#define blocky 16

__global__ void _weightBottomkernel_cuda(int num_slices, int num_grid, float beta, float *dev_F, float *dev_G, float* dev_wg5, float *dev_recon)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int n = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if ((k >= num_slices) || (n < 1) || (n >= (num_grid - 1)))
        return;

    // Pre-calculate the common indices
    int num_grid_sq = num_grid * num_grid;
    int ind0 = n + (num_grid - 1) * num_grid + k * num_grid_sq;

    int indg[5];
    indg[0] = ind0 + 1;
    indg[1] = ind0 - 1;
    indg[2] = ind0 - num_grid;
    indg[3] = ind0 - num_grid + 1;
    indg[4] = ind0 - num_grid - 1;

    // Utilize shared memory to optimize access
    float wg5_shared[5];
    #pragma unroll
    for (int q = 0; q < 5; q++) {
        wg5_shared[q] = 2 * beta * dev_wg5[q];
    }

    // Combine updates into a single loop
    float f_update = 0.0f;
    float g_update = 0.0f;
    #pragma unroll
    for (int q = 0; q < 5; q++) {
        f_update += wg5_shared[q];
        g_update -= wg5_shared[q] * (dev_recon[ind0] + dev_recon[indg[q]]);
    }
    
    // Update global memory at once
    dev_F[ind0] += f_update;
    dev_G[ind0] += g_update;
}