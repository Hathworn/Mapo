#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
Copyright 2014-2015 Dake Feng, Peri LLC, dakefeng@gmail.com

This file is part of TomograPeri.

TomograPeri is free software: you can redistribute it and/or modify
it under the terms of the GNU Lesser General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

TomograPeri is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU Lesser General Public License
along with TomograPeri.  If not, see <http://www.gnu.org/licenses/>.
*/

#define blockx 16
#define blocky 16

__global__ void _GEFrkernel_cuda(int num_slices, int num_grid, float* dev_recon, float* dev_G, float* dev_E, float* dev_F)
{
    uint m = blockIdx.x * blockDim.x + threadIdx.x;
    uint n = blockIdx.y * blockDim.y + threadIdx.y;
    uint k = blockIdx.z;

    // Improved index calculation using shared memory
    extern __shared__ float shared[];
    unsigned int grid_size = num_grid * num_grid * num_slices;
    if (m < num_grid && n < num_grid && k < num_slices) {
        uint i = m + n * num_grid + k * num_grid * num_grid;

        // Use shared memory to optimize memory access
        shared[threadIdx.x + threadIdx.y * blockDim.x] = dev_G[i];
        __syncthreads();

        float G_val = shared[threadIdx.x + threadIdx.y * blockDim.x];
        float E_val = dev_E[i];
        float F_val = dev_F[i];
        
        dev_recon[i] = (-G_val + sqrtf(G_val * G_val - 8.0f * E_val * F_val)) / (4.0f * F_val);
    }
}