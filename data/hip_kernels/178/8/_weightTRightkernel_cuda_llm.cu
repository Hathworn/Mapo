#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
Copyright 2014-2015 Dake Feng, Peri LLC, dakefeng@gmail.com

This file is part of TomograPeri.

TomograPeri is free software: you can redistribute it and/or modify
it under the terms of the GNU Lesser General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

TomograPeri is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU Lesser General Public License
along with TomograPeri.  If not, see <http://www.gnu.org/licenses/>.
*/

#define blockx 16
#define blocky 16

__global__ void _weightTRightkernel_cuda(int num_slices, int num_grid, float beta, float * __restrict__ dev_F, const float * __restrict__ dev_G, const float* __restrict__ dev_wg3, const float *__restrict__ dev_recon)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k >= num_slices)
        return;

    // Calculate initial index positions
    int ind0 = (num_grid - 1) + k * num_grid * num_grid;
    int indg[3];
    indg[0] = ind0 - 1;
    indg[1] = ind0 + num_grid;
    indg[2] = ind0 + num_grid - 1;

    float beta2 = 2.0f * beta;  // Precompute constant multiplier
    float dev_F_shared = dev_F[ind0];  // Load dev_F[ind0] once

    for (int q = 0; q < 3; q++) {
        float wg3_val = dev_wg3[q];
        dev_F_shared += beta2 * wg3_val;  // Use shared value for dev_F
        dev_G[ind0] -= beta2 * wg3_val * (dev_recon[ind0] + dev_recon[indg[q]]);
    }

    dev_F[ind0] = dev_F_shared;  // Store back the computed result
}