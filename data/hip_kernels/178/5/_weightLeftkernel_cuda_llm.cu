#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define blockx 16
#define blocky 16

__global__ void _weightLeftkernel_cuda(int num_slices, int num_grid, float beta, float *dev_F, float *dev_G, float* dev_wg5, float *dev_recon)
{
    // Calculate thread indices
    uint k = blockIdx.x * blockDim.x + threadIdx.x;
    uint n = blockIdx.y * blockDim.y + threadIdx.y + 1;

    // Boundary check
    if ((k >= num_slices) || (n < 1) || (n >= (num_grid - 1)))
        return;

    int ind0 = n * num_grid + k * num_grid * num_grid;

    // Compute indg indices
    int indg[5];
    indg[0] = ind0 + 1;
    indg[1] = ind0 + num_grid;
    indg[2] = ind0 - num_grid;
    indg[3] = ind0 + num_grid + 1;
    indg[4] = ind0 - num_grid + 1;

    // Loop unrolling for optimization
    #pragma unroll
    for (int q = 0; q < 5; q++) {
        float wg5q = 2 * beta * dev_wg5[q];
        dev_F[ind0] += wg5q;
        dev_G[ind0] -= wg5q * (dev_recon[ind0] + dev_recon[indg[q]]);
    }
}