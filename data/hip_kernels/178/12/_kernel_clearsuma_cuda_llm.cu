#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define blockx 16
#define blocky 16

__global__ void _kernel_clearsuma_cuda(int num_grid, float *dev_suma)
{
    // Calculate global thread index
    uint m = blockIdx.x * blockDim.x + threadIdx.x;
    uint n = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within grid bounds
    if (m < num_grid && n < num_grid) {
        uint i = m + n * num_grid;
        // Set suma to 0 only for valid elements
        dev_suma[i] = 0.0;
    }
}