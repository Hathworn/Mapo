#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define blockx 16
#define blocky 16

__global__ void _weightBLeftkernel_cuda(int num_slices, int num_grid, float beta, float *dev_F, float *dev_G, float* dev_wg3, float *dev_recon)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k >= num_slices) return;

    int ind0 = (num_grid - 1) * num_grid + k * num_grid * num_grid;
    int indg[3] = { ind0 + 1, ind0 - num_grid, ind0 - num_grid + 1 };

    // Optimize redundant calculations
    float recon_ind0 = dev_recon[ind0];
    float beta_wg3[3] = { 2 * beta * dev_wg3[0], 2 * beta * dev_wg3[1], 2 * beta * dev_wg3[2] };

    for (int q = 0; q < 3; q++) {
        dev_F[ind0] += beta_wg3[q];
        dev_G[ind0] -= beta_wg3[q] * (recon_ind0 + dev_recon[indg[q]]);
    }
}