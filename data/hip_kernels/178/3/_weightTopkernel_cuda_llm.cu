#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define blockx 16
#define blocky 16

__global__ void _weightTopkernel_cuda(int num_slices, int num_grid, float beta, float *dev_F, float *dev_G, float *dev_wg5, float *dev_recon)
{
    int indg[5];
    uint k = blockIdx.x * blockDim.x + threadIdx.x;
    uint n = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if ((k >= num_slices) || (n < 1) || (n >= (num_grid - 1)))
        return;
    
    int ind0 = n + k * num_grid * num_grid;

    // Precompute indices to reduce redundant calculations
    indg[0] = ind0 + 1;
    indg[1] = ind0 - 1;
    indg[2] = ind0 + num_grid;
    indg[3] = ind0 + num_grid + 1;
    indg[4] = ind0 + num_grid - 1;

    // Aggregate operations to minimize memory accesses
    float F_update = 0.0f;
    float G_update = 0.0f;

    #pragma unroll // Unroll loop for performance
    for (int q = 0; q < 5; q++) {
        F_update += 2 * beta * dev_wg5[q];
        G_update -= 2 * beta * dev_wg5[q] * (dev_recon[ind0] + dev_recon[indg[q]]);
    }

    dev_F[ind0] += F_update;
    dev_G[ind0] += G_update;
}