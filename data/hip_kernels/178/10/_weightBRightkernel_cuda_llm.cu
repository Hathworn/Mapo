#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define blockx 16
#define blocky 16

__global__ void _weightBRightkernel_cuda(int num_slices, int num_grid, float beta, float *dev_F, float *dev_G, float* dev_wg3, float *dev_recon)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k >= num_slices)
        return;

    // Precompute indices and steps to improve memory access patterns
    int grid_square = num_grid * num_grid;
    int ind0 = (num_grid - 1) + (num_grid - 1) * num_grid + k * grid_square;
    int indg[3] = { ind0 - 1, ind0 - num_grid, ind0 - num_grid - 1 };
    
    float beta_wg0 = 2 * beta * dev_wg3[0];
    float beta_wg1 = 2 * beta * dev_wg3[1];
    float beta_wg2 = 2 * beta * dev_wg3[2];
    float recon_val0 = dev_recon[ind0];
    
    dev_F[ind0] += beta_wg0 + beta_wg1 + beta_wg2;
    
    // Unrolling loop for efficiency
    dev_G[ind0] -= beta_wg0 * (recon_val0 + dev_recon[indg[0]]);
    dev_G[ind0] -= beta_wg1 * (recon_val0 + dev_recon[indg[1]]);
    dev_G[ind0] -= beta_wg2 * (recon_val0 + dev_recon[indg[2]]);
}