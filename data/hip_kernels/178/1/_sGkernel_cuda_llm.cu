#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
Copyright 2014-2015 Dake Feng, Peri LLC, dakefeng@gmail.com

This file is part of TomograPeri.

TomograPeri is free software: you can redistribute it and/or modify
it under the terms of the GNU Lesser General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

TomograPeri is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU Lesser General Public License
along with TomograPeri.  If not, see <http://www.gnu.org/licenses/>.
*/

#define blockx 16
#define blocky 16

__global__ void _sGkernel_cuda(int num_slices, int num_grid, float* dev_G, float *dev_suma)
{
    // Using threadIdx and blockIdx directly reduces register usage
    uint m = threadIdx.x + blockIdx.x * blockDim.x;
    uint n = threadIdx.y + blockIdx.y * blockDim.y;
    uint k = blockIdx.z;
    
    // Check bounds to avoid out of range memory access
    if (m < num_grid && n < num_grid && k < num_slices) 
    {
        uint i = m + n * num_grid + k * num_grid * num_grid;
        uint j = m + n * num_grid;
        dev_G[i] += dev_suma[j];
    }
}