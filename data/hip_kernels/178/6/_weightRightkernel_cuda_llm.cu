#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define blockx 16
#define blocky 16

__global__ void _weightRightkernel_cuda(int num_slices, int num_grid, float beta, float *dev_F, float *dev_G, float *dev_wg5, float *dev_recon)
{
    uint k = blockIdx.x * blockDim.x + threadIdx.x;
    uint n = blockIdx.y * blockDim.y + threadIdx.y + 1;
    
    // Early exit for out of bounds
    if (k >= num_slices || n >= (num_grid - 1)) return;

    int ind0 = (num_grid - 1) + n * num_grid + k * num_grid * num_grid;
    int indg[5];
    indg[0] = ind0 - 1;
    indg[1] = ind0 + num_grid;
    indg[2] = ind0 - num_grid;
    indg[3] = ind0 + num_grid - 1;
    indg[4] = ind0 - num_grid - 1;

    float beta_wg5[5];
    #pragma unroll // Unroll loop for performance
    for (int i = 0; i < 5; i++) {
        beta_wg5[i] = 2 * beta * dev_wg5[i];
    }

    float recon_sum = dev_recon[ind0]; // Cache
    #pragma unroll // Unroll loop for performance
    for (int q = 0; q < 5; q++) {
        dev_F[ind0] += beta_wg5[q];
        dev_G[ind0] -= beta_wg5[q] * (recon_sum + dev_recon[indg[q]]);
    }
}