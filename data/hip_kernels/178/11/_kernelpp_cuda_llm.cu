#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
Copyright 2014-2015 Dake Feng, Peri LLC, dakefeng@gmail.com

This file is part of TomograPeri.

TomograPeri is free software: you can redistribute it and/or modify
it under the terms of the GNU Lesser General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

TomograPeri is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU Lesser General Public License
along with TomograPeri.  If not, see <http://www.gnu.org/licenses/>.
*/

#define blockx 16
#define blocky 16

__global__ void _kernelpp_cuda(int num_projections, float mov, int num_pixels, int num_grid, int num_slices, float* dev_gridx, float* dev_gridy, float* dev_suma, float* dev_E, float* dev_data, float* dev_recon, float* dev_theta) {
    uint q = blockIdx.x * blockDim.x + threadIdx.x;
    uint m = blockIdx.y * blockDim.y + threadIdx.y;
    if ((m >= num_pixels) || (q >= num_projections)) return;

    const double PI = 3.141592653589793238462;
    bool quadrant = ((dev_theta[q] >= 0 && dev_theta[q] < PI/2) || (dev_theta[q] >= PI && dev_theta[q] < 3*PI/2));
    float sinq = sin(dev_theta[q]);
    float cosq = cos(dev_theta[q]);
    float xi = -1e6;
    float yi = -(num_pixels - 1) / 2.0f + m + mov;
    float srcx = xi * cosq - yi * sinq;
    float srcy = xi * sinq + yi * cosq;
    float detx = -xi * cosq - yi * sinq;
    float dety = -xi * sinq + yi * cosq;
    float slope = (srcy - dety) / (srcx - detx);
    float islope = 1.0f / slope;

    float coordx[MAX_NUM_GRID], coordy[MAX_NUM_GRID];
    for (int n = 0; n <= num_grid; n++) {
        coordx[n] = islope * (dev_gridy[n] - srcy) + srcx;
        coordy[n] = slope * (dev_gridx[n] - srcx) + srcy;
    }

    float ax[MAX_NUM_GRID], ay[MAX_NUM_GRID], bx[MAX_NUM_GRID], by[MAX_NUM_GRID];
    int alen = 0, blen = 0;
    for (int n = 0; n <= num_grid; n++) {
        if (coordx[n] > dev_gridx[0] && coordx[n] < dev_gridx[num_grid]) {
            ax[alen] = coordx[n];
            ay[alen] = dev_gridy[n];
            alen++;
        }
        if (coordy[n] > dev_gridy[0] && coordy[n] < dev_gridy[num_grid]) {
            bx[blen] = dev_gridx[n];
            by[blen] = coordy[n];
            blen++;
        }
    }

    float coorx[MAX_NUM_GRID*2], coory[MAX_NUM_GRID*2];
    int len = alen + blen, i = 0, j = 0, k = 0;
    if (quadrant) {
        while (i < alen && j < blen) {
            if (ax[i] < bx[j]) {
                coorx[k] = ax[i];
                coory[k] = ay[i];
                i++;
            } else {
                coorx[k] = bx[j];
                coory[k] = by[j];
                j++;
            }
            k++;
        }
        while (i < alen) {
            coorx[k] = ax[i];
            coory[k] = ay[i];
            i++;
            k++;
        }
        while (j < blen) {
            coorx[k] = bx[j];
            coory[k] = by[j];
            j++;
            k++;
        }
    } else {
        while (i < alen && j < blen) {
            if (ax[alen - 1 - i] < bx[j]) {
                coorx[k] = ax[alen - 1 - i];
                coory[k] = ay[alen - 1 - i];
                i++;
            } else {
                coorx[k] = bx[j];
                coory[k] = by[j];
                j++;
            }
            k++;
        }
        while (i < alen) {
            coorx[k] = ax[alen - 1 - i];
            coory[k] = ay[alen - 1 - i];
            i++;
            k++;
        }
        while (j < blen) {
            coorx[k] = bx[j];
            coory[k] = by[j];
            j++;
            k++;
        }
    }

    float leng[MAX_NUM_GRID*2];
    int indi[MAX_NUM_GRID*2];
    for (int n = 0; n < len - 1; n++) {
        float diffx = coorx[n + 1] - coorx[n];
        float diffy = coory[n + 1] - coory[n];
        leng[n] = sqrt(diffx * diffx + diffy * diffy);
        float midx = (coorx[n + 1] + coorx[n]) / 2.0f;
        float midy = (coory[n + 1] + coory[n]) / 2.0f;
        int indx = (int)(midx + num_grid / 2.0f) - ((midx + num_grid / 2.0f) >= (int)(midx + num_grid / 2.0f));
        int indy = (int)(midy + num_grid / 2.0f) - ((midy + num_grid / 2.0f) >= (int)(midy + num_grid / 2.0f));
        indi[n] = indx + indy * num_grid;
    }

    for (int n = 0; n < len - 1; n++) {
        atomicAdd(&(dev_suma[indi[n]]), leng[n]);
    }

    for (int k = 0; k < num_slices; k++) {
        int i = k * num_grid * num_grid;
        int io = m + k * num_pixels + q * num_slices * num_pixels;

        float simdata = 0.0f;
        for (int n = 0; n < len - 1; n++) {
            simdata += dev_recon[indi[n] + i] * leng[n];
        }
        float upd = dev_data[io] / simdata;
        for (int n = 0; n < len - 1; n++) {
            atomicAdd(&(dev_E[indi[n] + i]), -dev_recon[indi[n] + i] * upd * leng[n]);
        }
    }
}