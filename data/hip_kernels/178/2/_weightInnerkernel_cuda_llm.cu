#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define blockx 16
#define blocky 16

__global__ void _weightInnerkernel_cuda(int num_slices, int num_grid, float beta, float *dev_F, float *dev_G, float *dev_wg8, float *dev_recon)
{
    uint m = blockIdx.x * blockDim.x + threadIdx.x + 1;
    uint n = blockIdx.y * blockDim.y + threadIdx.y + 1;
    uint k = blockIdx.z;

    if ((k >= num_slices) || (n < 1) || (n >= (num_grid - 1)) || (m < 1) || (m >= (num_grid - 1)))
        return;

    int ind0 = m + n * num_grid + k * num_grid * num_grid;
    int indg[8];

    // Pre-calculate indices to avoid repeated computation in the loop
    indg[0] = ind0 + 1;
    indg[1] = ind0 - 1;
    indg[2] = ind0 + num_grid;
    indg[3] = ind0 - num_grid;
    indg[4] = ind0 + num_grid + 1;
    indg[5] = ind0 + num_grid - 1;
    indg[6] = ind0 - num_grid + 1;
    indg[7] = ind0 - num_grid - 1;

    // Use temporary variables to minimize repeated memory access
    float tempF = dev_F[ind0];
    float tempG = dev_G[ind0];
    float reconInd0 = dev_recon[ind0];

    for (int q = 0; q < 8; q++) {
        float wg8_q = dev_wg8[q];
        float term = 2 * beta * wg8_q;
        float reconSum = reconInd0 + dev_recon[indg[q]];

        tempF += term;
        tempG -= term * reconSum;
    }

    // Write back to global memory
    dev_F[ind0] = tempF;
    dev_G[ind0] = tempG;
}