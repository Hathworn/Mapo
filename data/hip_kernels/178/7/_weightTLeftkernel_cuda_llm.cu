#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define blockx 16
#define blocky 16

__global__ void _weightTLeftkernel_cuda(int num_slices, int num_grid, float beta, float *dev_F, float *dev_G, float *dev_wg3, float *dev_recon)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (k >= num_slices) return;

    int ind0 = k * num_grid * num_grid;
    int indg[3] = {ind0 + 1, ind0 + num_grid, ind0 + num_grid + 1};

    float shared_wg3[3];
    #pragma unroll  // Unroll the loop for performance
    for (int q = 0; q < 3; ++q) {
        shared_wg3[q] = dev_wg3[q];
        dev_F[ind0] += 2 * beta * shared_wg3[q];
        dev_G[ind0] -= 2 * beta * shared_wg3[q] * (dev_recon[ind0] + dev_recon[indg[q]]);
    }
}