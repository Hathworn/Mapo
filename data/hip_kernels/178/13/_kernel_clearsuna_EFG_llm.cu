#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define blockx 16
#define blocky 16

__global__ void _kernel_clearsuna_EFG(int num_slices, int num_grid, float* dev_EFG)
{
    // Compute flattened thread index for better memory coalescing
    uint index = blockIdx.x * blockDim.x + threadIdx.x 
               + (blockIdx.y * blockDim.y + threadIdx.y) * num_grid 
               + blockIdx.z * num_grid * num_grid;

    // Only proceed if the index is within the bounds
    if (index < (num_slices * num_grid * num_grid)) {
        dev_EFG[index] = 0.0;
    }
}