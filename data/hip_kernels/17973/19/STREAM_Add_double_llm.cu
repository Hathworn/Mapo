#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Add_double(double *a, double *b, double *c, size_t len)
{
    // Calculate the global index of the thread
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use stride looping to process more elements per thread and reduce global memory access
    size_t stride = blockDim.x * gridDim.x;
    for (; idx < len; idx += stride) {
        c[idx] = a[idx] + b[idx];
    }
}