#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Scale_double(double *a, double *b, double scale, size_t len)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure idx is within the bounds of the array
    if (idx < len) {
        b[idx] = scale * a[idx];

        // Unrolling loop to improve memory access patterns
        size_t stride = blockDim.x * gridDim.x;
        for (idx += stride; idx < len; idx += stride) {
            b[idx] = scale * a[idx];
        }
    }
}