#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Scale(float *a, float *b, float scale, size_t len)
{
    // Use shared memory to improve data locality and cache performance
    extern __shared__ float shared_a[]; 

    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (idx < len) {
        // Load elements to shared memory
        shared_a[threadIdx.x] = a[idx];
        __syncthreads();
        
        // Perform the scaling operation using shared memory
        b[idx] = scale * shared_a[threadIdx.x];
    }
}