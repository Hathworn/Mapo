#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_array_double(double *a, double value, size_t len)
{
    // Calculate the global index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Optimize by using grid-stride loop for better performance
    for (size_t stride = blockDim.x * gridDim.x; idx < len; idx += stride) {
        a[idx] = value;
    }
}