#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Copy_double(double *a, double *b, size_t len)
{
    // Calculate the index for this thread
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Use shared memory to avoid repeated global memory access
    __shared__ double sharedMem[256];

    // Directly copy data using shared memory for better cache efficiency
    while (idx < len) {
        sharedMem[threadIdx.x] = a[idx];
        b[idx] = sharedMem[threadIdx.x];

        // Move to the next index for this thread
        idx += blockDim.x * gridDim.x;
    }
}