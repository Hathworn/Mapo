#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Add_Optimized(float *a, float *b, float *c, size_t len)
{
    // Calculate global thread index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Use more efficient branching to ensure valid index within bounds
    if (idx < len) 
    {
        // Perform vector addition for valid indices
        c[idx] = a[idx] + b[idx];
    }
}