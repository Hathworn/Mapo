#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Triad_Optimized(float *a, float *b, float *c, float scalar, size_t len)
{
    // Calculate the thread's global index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Improved loop: Iterate over multiple elements per thread to increase occupancy
    for (size_t i = idx; i < len; i += blockDim.x * gridDim.x) {
        c[i] = a[i] + scalar * b[i];
    }
}