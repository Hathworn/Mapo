#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Triad_Optimized_double(double *a, double *b, double *c, double scalar, size_t len)
{
    // Calculate global thread index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Loop over the indices from this starting point
    for (; idx < len; idx += blockDim.x * gridDim.x) {
        c[idx] = a[idx] + scalar * b[idx];
    }
}