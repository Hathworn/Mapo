#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Scale_Optimized_double(double *a, double *b, double scale, size_t len)
{
    // Calculate global thread index once using optimized expression.
    size_t idx = (blockDim.x * blockIdx.x) + threadIdx.x;

    // Use a loop to process multiple elements per thread for better occupancy.
    for (; idx < len; idx += blockDim.x * gridDim.x) {
        b[idx] = scale * a[idx];
    }
}