#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Add(float *a, float *b, float *c, size_t len)
{
    // Use shared memory for efficient memory access
    extern __shared__ float shared_memory[];
    
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;

    while (idx < len) {
        // Load data into shared memory
        shared_memory[threadIdx.x] = a[idx] + b[idx];

        // Ensure all threads have written to shared memory
        __syncthreads();
        
        // Write result from shared memory to global memory
        c[idx] = shared_memory[threadIdx.x];
        
        idx += stride;
    }
}