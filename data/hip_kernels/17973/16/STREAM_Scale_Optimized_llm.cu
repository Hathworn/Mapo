#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Scale_Optimized(float *a, float *b, float scale, size_t len)
{
    // Use shared memory to improve memory access efficiency
    extern __shared__ float shared_a[];

    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t tid = threadIdx.x;

    // Boundary check to prevent out-of-bounds access
    if (idx < len) {
        // Load data into shared memory
        shared_a[tid] = a[idx];
        __syncthreads();

        // Perform operation using shared memory
        b[idx] = scale * shared_a[tid];
    }
}