#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Copy_Optimized(float *a, float *b, size_t len)
{
    // Calculate the global thread index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use a strided approach for better memory coalescing and load balancing
    for (size_t i = idx; i < len; i += blockDim.x * gridDim.x) {
        b[i] = a[i];
    }
}