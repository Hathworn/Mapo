#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Triad(float *a, float *b, float *c, float scalar, size_t len)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;
    
    for (; idx < len; idx += stride) {
        // Compute c[idx] using fused multiply-add (FMA) for potential performance boost
        c[idx] = fmaf(scalar, b[idx], a[idx]);
    }
}