#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Copy_Optimized_double(double *a, double *b, size_t len)
{
    // Calculate global thread index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure thread operates within bounds
    if (idx < len) {
        b[idx] = a[idx];
    }
}