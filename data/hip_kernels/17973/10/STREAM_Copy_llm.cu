#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Copy(float *a, float *b, size_t len)
{
    // Use restrict to enable compiler optimizations 
    float* __restrict__ a_ptr = a;
    float* __restrict__ b_ptr = b;

    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Unroll loop for improved throughput
    for (; idx + 3 < len; idx += blockDim.x * gridDim.x * 4) {
        b_ptr[idx] = a_ptr[idx];
        b_ptr[idx + 1] = a_ptr[idx + 1];
        b_ptr[idx + 2] = a_ptr[idx + 2];
        b_ptr[idx + 3] = a_ptr[idx + 3];
    }

    // Handle remaining elements
    while (idx < len) {
        b_ptr[idx] = a_ptr[idx];
        idx += blockDim.x * gridDim.x;
    }
}