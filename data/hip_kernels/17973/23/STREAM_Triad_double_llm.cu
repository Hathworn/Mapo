#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Triad_double(double *a, double *b, double *c, double scalar, size_t len)
{
    // Calculate the linear thread index across the entire grid
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop for accessing all elements
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = idx; i < len; i += stride) {
        c[i] = a[i] + scalar * b[i];
    }
}