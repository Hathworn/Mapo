#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Add_Optimized_double(double *a, double *b, double *c, size_t len)
{
    // Calculate global index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Loop through elements in steps to process multiple in one go
    for (size_t i = idx; i < len; i += blockDim.x * gridDim.x)
    {
        // Add elements
        c[i] = a[i] + b[i];
    }
}