#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAddKernel(float* A, float* B, float* C, int n) {
    // Calculate global thread index -------------------
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Process only threads with indices less than n ----
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}