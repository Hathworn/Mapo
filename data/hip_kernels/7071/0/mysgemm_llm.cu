#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SZ 16

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

/********************************************************************
*
* Compute C = A x B
*   where A is a (m x k) matrix
*   where B is a (k x n) matrix
*   where C is a (m x n) matrix
*
* Use shared memory for tiling
*
********************************************************************/

unsigned int TiRow = threadIdx.y;
unsigned int TiCol = threadIdx.x;
unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

__shared__ float As[TILE_SZ][TILE_SZ];
__shared__ float Bs[TILE_SZ][TILE_SZ];

float sum = 0.0f;

for(unsigned int TiNum = 0; TiNum < (k+TILE_SZ-1)/TILE_SZ; TiNum++){
    // Load tile from A
    if(row < m && (TiNum * TILE_SZ + TiCol) < k)
        As[TiRow][TiCol] = A[row * k + TiNum * TILE_SZ + TiCol];
    else
        As[TiRow][TiCol] = 0.0f;

    // Load tile from B
    if((TiNum * TILE_SZ + TiRow) < k && col < n)
        Bs[TiRow][TiCol] = B[(TiNum * TILE_SZ + TiRow) * n + col];
    else
        Bs[TiRow][TiCol] = 0.0f;

    __syncthreads();

    // Compute tile multiplication
    for(unsigned int TiElem = 0; TiElem < TILE_SZ; ++TiElem)
        sum += As[TiRow][TiElem] * Bs[TiElem][TiCol];

    __syncthreads();
}

// Store result in C
if (row < m && col < n)
    C[row * n + col] = sum;
}