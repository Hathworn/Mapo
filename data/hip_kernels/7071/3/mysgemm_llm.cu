#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

/********************************************************************
*
* Compute C = A x B
*   where A is a (m x k) matrix
*   where B is a (k x n) matrix
*   where C is a (m x n) matrix
*
********************************************************************/

// Calculate global row and column indices
int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;

// Check boundary conditions
if(row < m && col < n) {
    float acc = 0;

    // Use unrolling for the accumulation loop to improve performance
    int index;
    for(index = 0; index < (k - 4); index += 4) {
        acc += A[row * k + index] * B[index * n + col];
        acc += A[row * k + index + 1] * B[(index + 1) * n + col];
        acc += A[row * k + index + 2] * B[(index + 2) * n + col];
        acc += A[row * k + index + 3] * B[(index + 3) * n + col];
    }
    // Handle remaining iterations
    for(; index < k; index++) {
        acc += A[row * k + index] * B[index * n + col];
    }

    // Write the result
    C[row * n + col] = acc;
}
}