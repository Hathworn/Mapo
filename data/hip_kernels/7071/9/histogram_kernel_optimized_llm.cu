#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogram_kernel_optimized(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins) {
    // Use shared memory for bin storage
    extern __shared__ unsigned int bins_s[];

    // Initialize shared memory bins using parallel threads
    int thid = threadIdx.x;
    for (int i = thid; i < num_bins; i += blockDim.x) {
        bins_s[i] = 0u;
    }
    __syncthreads();

    // Efficient histogram calculation using per-thread accumulation
    unsigned int element = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int accumulator = 0;
    unsigned int prev_index = 0;

    while (element < num_elements) {
        unsigned int curr_index = input[element];

        if (curr_index != prev_index) {
            atomicAdd(&(bins_s[prev_index]), accumulator);
            accumulator = 1;
            prev_index = curr_index;
        } else {
            accumulator++;
        }
        element += blockDim.x * gridDim.x;
    }

    // Add remaining accumulation
    if (accumulator > 0) {
        atomicAdd(&(bins_s[prev_index]), accumulator);
    }
    __syncthreads();

    // Transfer results from shared memory bins to global memory bins
    for (int i = thid; i < num_bins; i += blockDim.x) {
        atomicAdd(&(bins[i]), bins_s[i]);
    }
}