#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAddKernel(float* A, float* B, float* C, int n) {
    // Calculate global thread index based on the block and thread indices
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Use global index to determine which elements to read, add, and write
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}