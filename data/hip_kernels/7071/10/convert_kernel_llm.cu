#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convert_kernel(unsigned int *bins32, uint8_t *bins8, unsigned int num_bins) {
    // Calculate the global thread index
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop over elements assigned to the current thread
    for (int i = thid; i < num_bins; i += blockDim.x * gridDim.x) {
        // Use local register to reduce global memory access
        unsigned int reg_bin = bins32[i];
        
        // Standard conditional operation
        bins8[i] = (reg_bin > 255) ? 255u : (uint8_t)reg_bin;
    }
}