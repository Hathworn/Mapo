#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogram_kernel(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins) {
    extern __shared__ unsigned int bins_s[];

    int thid = threadIdx.x;
    // Initialize shared memory bins to zero
    for (int i = thid; i < num_bins; i += blockDim.x) {
        bins_s[i] = 0u;
    }
    __syncthreads();

    // Histogram calculation in shared memory
    for (int i = blockIdx.x * blockDim.x + thid; i < num_elements; i += blockDim.x * gridDim.x) {
        atomicAdd(&(bins_s[input[i]]), 1);
    }
    __syncthreads();

    // Accumulate the shared memory bins into global memory
    for (int i = thid; i < num_bins; i += blockDim.x) {
        atomicAdd(&(bins[i]), bins_s[i]);
    }
}