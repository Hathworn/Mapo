#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Stride loop to handle larger arrays
    for (int i = idx; i < N; i += blockDim.x * gridDim.x) {
        X[i*INCX] *= ALPHA;
    }
}