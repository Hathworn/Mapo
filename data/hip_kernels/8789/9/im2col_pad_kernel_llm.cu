#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel function
__global__ void im2col_pad_kernel(float *im, int channels, int height, int width, int ksize, int stride, float *data_col)
{
    int height_col = 1 + (height-1) / stride;
    int width_col = 1 + (width-1) / stride;
    int channels_col = channels * ksize * ksize;
    int pad = ksize / 2;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int col_size = height_col * width_col * channels_col;
    if (id >= col_size) return;

    int w = id % width_col;
    int h = (id / width_col) % height_col;
    int c = (id / width_col / height_col) % channels_col;

    int w_offset = c % ksize;
    int h_offset = c / ksize % ksize;
    int im_channel = c / (ksize * ksize);
    int im_row = h_offset + h * stride - pad;
    int im_col = w_offset + w * stride - pad;

    // Using conditional operator for compact readability
    data_col[id] = (im_row < 0 || im_col < 0 || im_row >= height || im_col >= width) ? 0 : im[im_col + width * (im_row + height * im_channel)];
}