#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize grid index calculation
    if (i < groups) { // Reorder conditions to improve readability
        float temp_sum = 0.0f; // Use register for temporary sum
        for (int k = 0; k < n; ++k) {
            temp_sum += x[k * groups + i];
        }
        sum[i] = temp_sum; // Write back to global memory
    }
}