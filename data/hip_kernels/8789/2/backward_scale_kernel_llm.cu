#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0.0f;

    // Unroll loop for better performance
    for (int b = 0; b < batch; ++b) {
        int base_idx = size * (filter + n * b);
        for (int i = p; i < size; i += BLOCK) {
            int index = i + base_idx;
            sum += delta[index] * x_norm[index];
        }
    }

    part[p] = sum;
    __syncthreads();

    // Use parallel reduction for better performance
    if (p < BLOCK / 2) {
        part[p] += part[p + BLOCK / 2];
    }
    __syncthreads();
    
    if (p < BLOCK / 4) {
        part[p] += part[p + BLOCK / 4];
    }
    __syncthreads();
    
    if (p < BLOCK / 8) {
        part[p] += part[p + BLOCK / 8];
    }
    __syncthreads();
    
    if (p < BLOCK / 16) {
        part[p] += part[p + BLOCK / 16];
    }
    __syncthreads();
    
    if (p == 0) {
        scale_updates[filter] += part[0];
    }
}