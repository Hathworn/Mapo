#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Optimize memory access and loop iteration using a single active warp per block
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    int stride = blockDim.x * gridDim.x; // Stride to cover all elements

    for (int i = idx; i < N; i += stride) {
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
    }
}