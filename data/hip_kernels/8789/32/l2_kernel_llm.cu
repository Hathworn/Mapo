#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate unique thread index for the global array
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process only valid indices
    if(i < n){
        float diff = truth[i] - pred[i];
        
        // Compute squared error and store result
        error[i] = __fmul_rn(diff, diff);  // Use efficient floating-point multiplication
        
        // Store the difference for delta
        delta[i] = diff;
    }
}