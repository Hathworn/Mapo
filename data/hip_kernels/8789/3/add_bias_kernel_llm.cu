#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    // Compute the global thread index
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Check if the thread is within bounds and perform addition
    if(offset < size) 
    {
        int index = (batch * n + filter) * size + offset;
        output[index] += biases[filter];
    }
}