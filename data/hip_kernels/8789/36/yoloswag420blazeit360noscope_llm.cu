#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified block index calculation
    if (id < size) {
        float r = rand[id]; // Cache rand value to avoid repeated global memory access
        input[id] = (r < prob) ? 0.0f : input[id] * scale;
    }
}