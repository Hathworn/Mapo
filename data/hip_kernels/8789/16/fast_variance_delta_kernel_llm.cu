#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void  fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0.0f;

    int filter = blockIdx.x;
    int thread_stride = blockDim.x * gridDim.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = id; i < spatial; i += thread_stride){
            int index = j * spatial * filters + filter * spatial + i;

            if (i < spatial) {
                local[id] += delta[index] * (x[index] - mean[filter]);
            }
        }
    }
    __syncthreads(); // Ensure all threads have written to local

    // Use reduction to accumulate results
    for (int stride = threads / 2; stride > 0; stride >>= 1) {
        if (id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads(); // Synchronize threads after each reduction step
    }

    if (id == 0) {
        variance_delta[filter] = local[0] * -.5f * powf(variance[filter] + .000001f, -1.5f);
    }
}