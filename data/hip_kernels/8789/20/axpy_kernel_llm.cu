#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation
    if(i < N) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX]; // Use consistent spacing for better readability
    }
}