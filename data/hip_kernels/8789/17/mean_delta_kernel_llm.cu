#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within the filter range
    if (i >= filters) return;

    // Local accumulator to reduce memory accesses
    float local_mean_delta = 0.0f;
    for (int j = 0; j < batch; ++j) {
        int base_index = j * filters * spatial + i * spatial;
        for (int k = 0; k < spatial; ++k) {
            local_mean_delta += delta[base_index + k];
        }
    }

    // Use a local variable to store the calculation result
    mean_delta[i] = local_mean_delta * (-1.0f / sqrtf(variance[i] + .000001f));
}