#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate unique thread index globally
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Parallel iteration over elements handled by the thread with a loop
    for (int i = idx; i < N; i += blockDim.x * gridDim.x)
    {
        // Set X[i*INCX] to ALPHA
        X[i * INCX] = ALPHA;
    }
}