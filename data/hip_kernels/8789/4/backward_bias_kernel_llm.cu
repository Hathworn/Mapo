#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 256

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size) {
    __shared__ float part[BLOCK_SIZE];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0.0f;

    // Unroll loop and use warp-level reduction for performance optimization
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK_SIZE) {
            int index = i + size * (filter + n * b);
            sum += delta[index];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Intra-block reduction
    if (p < 128) part[p] += part[p + 128];
    __syncthreads();
    if (p < 64) part[p] += part[p + 64];
    __syncthreads();
    if (p < 32) {
        part[p] += part[p + 32];
        part[p] += part[p + 16];
        part[p] += part[p + 8];
        part[p] += part[p + 4];
        part[p] += part[p + 2];
        part[p] += part[p + 1];
    }

    if (p == 0) {
        // Atomic add to ensure correctness without race conditions
        atomicAdd(&bias_updates[filter], part[0]);
    }
}