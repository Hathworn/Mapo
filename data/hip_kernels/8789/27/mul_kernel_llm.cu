#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop to cover all elements if N > total number of threads
    for (; i < N; i += blockDim.x * gridDim.x)
    {
        Y[i * INCY] *= X[i * INCX]; // Perform multiplication
    }
}