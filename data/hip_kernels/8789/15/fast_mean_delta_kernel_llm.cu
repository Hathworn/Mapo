#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    int spatialPerThread = (spatial + threads - 1) / threads;  // Ensure all spatial indices are handled

    local[id] = 0;

    // Optimized loop to utilize shared memory coalescing
    for(int j = 0; j < batch; ++j){
        for(int i = id; i < spatial; i += threads){
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += delta[index];
        }
    }

    __syncthreads(); // Ensure all threads have completed summing

    // Use a single thread per block to sum the local sums and compute mean_delta
    if (id == 0){
        mean_delta[filter] = 0;
        for (int i = 0; i < threads; ++i){
            mean_delta[filter] += local[i];
        }
        mean_delta[filter] *= (-1.0f / sqrtf(variance[filter] + .000001f));
    }
}
```
