#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, float *input, float *output, int *indexes)
{
    int h = (in_h-1)/stride + 1;
    int w = (in_w-1)/stride + 1;
    int c = in_c;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = (-size-1)/2 + 1;
    int h_offset = (-size-1)/2 + 1;

    int out_index = j + w*(i + h*(k + c*b));
    float max = -INFINITY;
    int max_i = -1;

    // Pre-compute for efficient access
    int base_h = h_offset + i * stride;
    int base_w = w_offset + j * stride;

    // Iterate over the pooling window
    for(int l = 0; l < size; ++l){
        int cur_h = base_h + l;
        if(cur_h < 0 || cur_h >= in_h) continue; // Skip out of bounds

        for(int m = 0; m < size; ++m){
            int cur_w = base_w + m;
            if(cur_w < 0 || cur_w >= in_w) continue; // Skip out of bounds

            int index = cur_w + in_w*(cur_h + in_h*(k + b*in_c));
            float val = input[index];
            if (val > max) {
                max = val;        // Update max
                max_i = index;    // Update index
            }
        }
    }

    output[out_index] = max;
    indexes[out_index] = max_i;
}