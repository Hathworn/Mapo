#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0.0f;

    int filter = blockIdx.x;

    // Optimized workload by using a single loop
    for(int j = 0; j < batch; ++j){
        for(int i = id; i < spatial; i += threads){
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += x[index];
        }
    }

    __syncthreads();  // Ensure all threads have completed local accumulation

    // Use parallel reduction for better performance
    if (threads > 1) {
        for (int s = threads / 2; s > 0; s >>= 1) {
            if (id < s) {
                local[id] += local[id + s];
            }
            __syncthreads();  // Synchronize threads after each reduction step
        }
    }

    if (id == 0) {
        mean[filter] = local[0] / (spatial * batch);
    }
}