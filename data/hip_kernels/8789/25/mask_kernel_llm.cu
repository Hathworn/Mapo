#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Use 1D grid
    if (i < n && mask[i] == mask_num) // Check bounds and condition
    {
        x[i] = mask_num; // Set x[i] to mask_num
    }
}