#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h-1)/stride + 1;
    int w = (in_w-1)/stride + 1;
    int c = in_c;
    int area = (size-1)/stride;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    // Calculate indices
    int index = id;
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    int w_offset = (-size-1)/2 + 1;
    int h_offset = (-size-1)/2 + 1;

    float d = 0;

    // Unroll the loops and use fewer integer arithmetic operations
    #pragma unroll
    for(int l = -area; l < area+1; ++l){
        int out_h = (i-h_offset)/stride + l;
        if(out_h < 0 || out_h >= h) continue; // Skip invalid rows
        #pragma unroll
        for(int m = -area; m < area+1; ++m){
            int out_w = (j-w_offset)/stride + m;
            if(out_w >= 0 && out_w < w) { // Check column validity
                // Calculate the output index only once
                int out_index = out_w + w*(out_h + h*(k + c*b));
                d += (indexes[out_index] == index) ? delta[out_index] : 0;
            }
        }
    }
    prev_delta[index] += d;
}