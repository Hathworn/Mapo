#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified grid and block indexing
    if (id >= size) return;

    // Precompute divisors to reduce recalculation
    int size_minw = size / minw;
    int size_minh = size_minw / minh;
    int size_minc = size_minh / minc;

    int b = id / size_minc;
    id %= size_minc;
    int k = id / size_minh;
    id %= size_minh;
    int j = id / minw;
    int i = id % minw;

    int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
    int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));
    atomicAdd(&out[out_index], add[add_index]); // Use atomicAdd for potential race condition
}