#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_softmax_layer_kernel(int n, int batch, float *input, float temp, float *output)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;  // Use simpler block index calculation
    if (b >= batch) return;

    float sum = 0;
    float largest = -INFINITY;

    // Use shared memory for storing intermediate results to reduce global memory access
    extern __shared__ float shared_input[];
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        shared_input[i] = input[i + b * n];
    }
    __syncthreads();

    // Find the largest value
    for (int i = 0; i < n; ++i) {
        largest = fmaxf(shared_input[i], largest);
    }

    // Compute exponentials and sum them
    for (int i = 0; i < n; ++i) {
        sum += expf((shared_input[i] / temp) - (largest / temp));
    }

    sum = (sum != 0) ? (largest/temp + logf(sum)) : (largest - 100);

    // Compute final output values
    for (int i = 0; i < n; ++i) {
        output[i + b * n] = expf((shared_input[i] / temp) - sum);
    }
}