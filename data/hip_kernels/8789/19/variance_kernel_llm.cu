#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Pre-compute scale factor outside of kernel loop for efficiency
    float scale = 1.0f / (batch * spatial - 1);
    
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use blockDim.y to distribute filters among multiple blocks if needed
    if (i >= filters) return;
    
    float var = 0.0f;
    
    // Combine nested loops to use single loop for better cache coherence
    for(int j = 0; j < batch * spatial; ++j) {
        int index = j * filters + i; // Directly access flattened index
        var += powf((x[index] - mean[i]), 2); // Utilize powf for single-precision float
    }
    
    variance[i] = var * scale;
}