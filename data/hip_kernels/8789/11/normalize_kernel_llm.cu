#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Loop to handle cases where N is larger than grid size
    for (; index < N; index += stride) {
        int f = (index / spatial) % filters;
        // Optimize division operation by using multiplication and addition
        float inv_sqrt_var = rsqrtf(variance[f] + .000001f);
        x[index] = (x[index] - mean[f]) * inv_sqrt_var;
    }
}