#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Cache variance[i] and mean[i] in registers for better performance
    float mean_i = mean[i];
    float variance_i = variance[i];
    float temp = 0.0f;

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            temp += delta[index] * (x[index] - mean_i);
        }
    }

    // Use pre-computed constant and avoid pow() for better performance
    float variance_pow = -.5f * __frcp_rn(sqrtf(variance_i + .000001f) * (variance_i + .000001f));
    variance_delta[i] = temp * variance_pow;
}