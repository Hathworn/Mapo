#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Define scale outside the loop as it's independent of the calculation.
    float scale = 1.0f / (batch * spatial);

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < filters) {
        float sum = 0.0f;
        for (int j = 0; j < batch; ++j) {
            for (int k = 0; k < spatial; ++k) {
                // Precompute index to avoid repeated multiplication.
                int index = j * filters * spatial + i * spatial + k;
                sum += x[index];
            }
        }
        // Store the result in one step to account for memory access patterns.
        mean[i] = sum * scale;
    }
}