#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int k = id % c;
    int b = id / c;

    float out_val = out_delta[k + c * b] / (w * h);  // Precompute division to reduce redundant operations
    int offset = h * w * (k + b * c);

    // Use a pointer to reduce repeated index computation
    float *in_ptr = in_delta + offset;

    for (int i = 0; i < w * h; ++i) {
        in_ptr[i] += out_val;
    }
}