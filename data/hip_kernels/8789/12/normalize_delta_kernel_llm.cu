#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    // Calculate unique global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we do not exceed bounds
    if (index >= N) return;

    // Calculate filter index
    int f = (index / spatial) % filters;

    // Use floating-point constant and common subexpression elimination to optimize
    float one_over_sqrt_var = 1.0f / (sqrtf(variance[f]) + 0.000001f);
    float two_over_spatial_batch = 2.0f / (spatial * batch);

    // Update delta using optimized expression
    delta[index] = delta[index] * one_over_sqrt_var + 
                   variance_delta[f] * two_over_spatial_batch * (x[index] - mean[f]) + 
                   mean_delta[f] / (spatial * batch);
}