#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    int b = id / c; // Simplified division

    int out_index = k + c * b;
    float sum = 0.0f; // Use local variable for accumulation
    int in_index_base = h * w * (k + b * c); // Precompute base index

    // Efficiently calculate sum using a single loop with precomputed index base
    for(int i = 0; i < w * h; ++i){
        sum += input[in_index_base + i];
    }

    output[out_index] = sum / (w * h); // Single division after accumulation
}