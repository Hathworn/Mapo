#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void im2col_nopad_kernel(float *im, int channels, int height, int width, int ksize, int stride, float *data_col)
{
    // Calculate col_size, early return if out of bounds
    int height_col = (height - ksize) / stride + 1;
    int width_col = (width - ksize) / stride + 1;
    int channels_col = channels * ksize * ksize;
    int col_size = height_col * width_col * channels_col;

    // Calculate unique global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= col_size) return;

    // Use alternative method for index calculation
    int c = id % channels_col;
    int h = (id / channels_col) % height_col;
    int w = (id / (channels_col * height_col));

    // Calculate offsets and indexes
    int w_offset = c % ksize;
    int h_offset = (c / ksize) % ksize;
    int im_channel = c / (ksize * ksize);
    int im_row = h_offset + h * stride;
    int im_col = w_offset + w * stride;

    // Check bounds and assign value
    int im_index = im_col + width * (im_row + height * im_channel);
    data_col[id] = (im_row < 0 || im_col < 0 || im_row >= height || im_col >= width) ? 0 : im[im_index];
}