#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_filters_kernel(float *filters, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x;
    if (f >= n) return;

    float mean = 0.0f;

    // Use shared memory to reduce global memory access
    extern __shared__ float shared_memory[];
    float *local_filters = shared_memory;

    // Load data into shared memory
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        local_filters[i] = abs(filters[f * size + i]);
    }
    __syncthreads();

    // Accumulate sum in shared memory
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        mean += local_filters[i];
    }
   __syncthreads();
   
    // Calculate the mean
    if (threadIdx.x == 0) {
        for (int i = 1; i < blockDim.x; i++) {
            mean += local_filters[i];
        }
        mean /= size;
    }
    __syncthreads();

    // Binarization
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (filters[f * size + i] > 0) ? mean : -mean;
    }
}