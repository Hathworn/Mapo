#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure we don't process out-of-bounds elements
    if(i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff); // Use fast math function
        if(abs_val < 1.0f) {
            error[i] = diff * diff;
            delta[i] = diff;
        }
        else {
            error[i] = abs_val * 2.0f - 1.0f;
            delta[i] = (diff < 0.0f) ? -1.0f : 1.0f;
        }
    }
}