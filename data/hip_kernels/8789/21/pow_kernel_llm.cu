#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Optimize index calculation using 2D grid mapping
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Use shared memory for temporary data storage (if applicable)
    if (i < N) {
        float x_val = X[i * INCX]; // Load once for reuse
        Y[i * INCY] = powf(x_val, ALPHA); // Use faster powf for float
    }
}