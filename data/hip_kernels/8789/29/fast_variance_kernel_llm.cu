#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0;

    float sum = 0;
    int maxSpatial = (spatial / threads) * threads;  

    for(int j = 0; j < batch; ++j){
        for(int i = id; i < maxSpatial; i += threads){
            int index = j * spatial * filters + filter * spatial + i;
            sum += pow(x[index] - mean[filter], 2);
        }
        if(id + maxSpatial < spatial) {  
            int index = j * spatial * filters + filter * spatial + id + maxSpatial;
            sum += pow(x[index] - mean[filter], 2);
        }
    }

    local[id] = sum;

    __syncthreads();  

    if (id == 0) {
        float variance_value = 0;
        for(int i = 0; i < threads; ++i){
            variance_value += local[i];
        }
        
        variance[filter] = variance_value / (spatial * batch - 1);
    }
}