#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int filter = blockIdx.y;
    int batch = blockIdx.z;
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop for better occupancy and performance
    for (int offset = threadId; offset < size; offset += blockDim.x * gridDim.x)
    {
        output[(batch * n + filter) * size + offset] *= biases[filter];
    }
}