#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Constants used by the program
#define BLOCK_DIM                      16
#define DEBUG                          0

__global__ void cuParallelSqrt(float *dist, int width, int k) {
    // Calculate linear thread index to optimize global memory access
    unsigned int index = blockIdx.y * blockDim.y * width + blockIdx.x * blockDim.x + threadIdx.y * width + threadIdx.x;
    
    if (index < k * width) {
        dist[index] = sqrt(dist[index]);
    }
}