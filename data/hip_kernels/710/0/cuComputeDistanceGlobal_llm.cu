#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Constants used by the program
#define BLOCK_DIM                      16

__global__ void cuComputeDistanceGlobal( float* A, int wA, float* B, int wB, int dim, float* AB) {

    __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Start positions in A and B
    int begin_A = BLOCK_DIM * blockIdx.y;
    int begin_B = BLOCK_DIM * blockIdx.x;
    int step_A = BLOCK_DIM * wA;
    int step_B = BLOCK_DIM * wB;
    int end_A = begin_A + (dim - 1) * wA;

    float ssd = 0.0f;

    int cond0 = (begin_A + tx < wA);
    int cond1 = (begin_B + tx < wB);
    int cond2 = (begin_A + ty < wA);

    // Iterate over sub-matrices
    for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {
        // Load elements into shared memory
        if (a / wA + ty < dim) {
            shared_A[ty][tx] = (cond0) ? A[a + wA * ty + tx] : 0.0f;
            shared_B[ty][tx] = (cond1) ? B[b + wB * ty + tx] : 0.0f;
        } else {
            shared_A[ty][tx] = 0.0f;
            shared_B[ty][tx] = 0.0f;
        }

        __syncthreads();

        // Compute block sub-matrix
        if (cond2 && cond1) {
            #pragma unroll  // Enable loop unrolling for better performance
            for (int k = 0; k < BLOCK_DIM; ++k) {
                float tmp = shared_A[k][ty] - shared_B[k][tx];
                ssd += tmp * tmp;
            }
        }

        __syncthreads();
    }
    
    // Write the results back to global memory
    if (cond2 && cond1)
        AB[(begin_A + ty) * wB + begin_B + tx] = ssd;
}