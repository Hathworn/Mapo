#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Constants used by the program
#define BLOCK_DIM                      16
#define DEBUG                          0

__global__ void cuInsertionSort(float *dist, long *ind, int width, int height, int k) {

    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (xIndex < width) {
        float *p_dist = dist + xIndex;
        long *p_ind = ind + xIndex;

        float max_dist = p_dist[0];
        p_ind[0] = 1;

        // Sort first k elements
        for (int l = 1; l < k; l++) {
            long curr_row = l * width;
            float curr_dist = p_dist[curr_row];
            if (curr_dist < max_dist) {
                int i = l - 1;
                while (i >= 0 && p_dist[i * width] > curr_dist) {
                    p_dist[(i + 1) * width] = p_dist[i * width];
                    p_ind[(i + 1) * width] = p_ind[i * width];
                    i--;
                }
                p_dist[(i + 1) * width] = curr_dist;
                p_ind[(i + 1) * width] = l + 1;
                max_dist = p_dist[(k - 1) * width];
            } else {
                p_ind[l * width] = l + 1;
            }
            max_dist = p_dist[l * width];
        }

        // Insert elements into k-th sorted list
        for (int l = k; l < height; l++) {
            float curr_dist = p_dist[l * width];
            if (curr_dist < max_dist) {
                int i = k - 1;
                while (i >= 0 && p_dist[i * width] > curr_dist) {
                    p_dist[(i + 1) * width] = p_dist[i * width];
                    p_ind[(i + 1) * width] = p_ind[i * width];
                    i--;
                }
                p_dist[(i + 1) * width] = curr_dist;
                p_ind[(i + 1) * width] = l + 1;
                max_dist = p_dist[(k - 1) * width];
            }
        }
    }
}