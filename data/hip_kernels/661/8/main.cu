#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "bilateralFilterCudaKernel.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float3 *dev_input = NULL;
hipMalloc(&dev_input, XSIZE*YSIZE);
float3 *dev_output = NULL;
hipMalloc(&dev_output, XSIZE*YSIZE);
float l2norm = 1;
int width = XSIZE;
int height = YSIZE;
int radius = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
bilateralFilterCudaKernel<<<gridBlock, threadBlock>>>(dev_input,dev_output,l2norm,width,height,radius);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
bilateralFilterCudaKernel<<<gridBlock, threadBlock>>>(dev_input,dev_output,l2norm,width,height,radius);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
bilateralFilterCudaKernel<<<gridBlock, threadBlock>>>(dev_input,dev_output,l2norm,width,height,radius);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}