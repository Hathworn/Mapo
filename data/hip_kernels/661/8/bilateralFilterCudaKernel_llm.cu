#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float3 addCuda(float3 a, float3 b) {
    return{ a.x + b.x, a.y + b.y, a.z + b.z };
}

__device__ inline float3 multiplyCuda(float a, float3 b) {
    return{ a * b.x, a * b.y, a * b.z };
}

__device__ inline float euclideanLenCuda(float3 a, float3 b, float d) {
    float mod = (b.x - a.x) * (b.x - a.x) + (b.y - a.y) * (b.y - a.y) + (b.z - a.z) * (b.z - a.z);
    return expf(-mod / (2.0f * d * d));
}

__global__ void bilateralFilterCudaKernel(float3* dev_input, float3* dev_output, float l2norm, int width, int height, int radius) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < width) && (y < height))
    {
        float sum = 0.0f;
        float3 t = { 0.f, 0.f, 0.f };
        float3 center = dev_input[y * width + x];
        int r = radius;

        // Use shared memory for domain distances to reduce repetitive access
        __shared__ float c_gaussian_shared[256]; // Example size, adjust based on actual usage

        float domainDist = 0.0f, colorDist = 0.0f, factor = 0.0f;

        for (int i = -r; i <= r; i++) {
            int crtY = min(max(y + i, 0), height - 1);

            for (int j = -r; j <= r; ++j) {
                int crtX = min(max(x + j, 0), width - 1);

                float3 curPix = dev_input[crtY * width + crtX];
                domainDist = c_gaussian_shared[r + i] * c_gaussian_shared[r + j];
                colorDist = euclideanLenCuda(curPix, center, l2norm);
                factor = domainDist * colorDist;
                sum += factor;
                t = addCuda(t, multiplyCuda(factor, curPix));
            }
        }

        dev_output[y * width + x] = multiplyCuda(1.f / sum, t);
    }
}