#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory for better performance
    __shared__ float shared_accuracy;
    
    // Load data into shared memory
    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy;
    }
    __syncthreads();
    
    // Perform division and store result back
    if (threadIdx.x == 0) {
        shared_accuracy /= N;
        *accuracy = shared_accuracy;
    }
}