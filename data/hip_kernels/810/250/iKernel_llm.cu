#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void iKernel(float *src, float *dst, int n)
{
    // Calculate global index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (idx < n) {
        // Perform the computation
        dst[idx] = src[idx] * 2.0f;
    }
}