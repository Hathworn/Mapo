#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysZeroCopyWithUVA(float *A, float *B, float *C, const int N)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for improved memory access
    __shared__ float sA[256];
    __shared__ float sB[256];

    if (idx < N) {
        // Load data into shared memory
        sA[threadIdx.x] = A[idx];
        sB[threadIdx.x] = B[idx];

        // Make sure all threads have loaded their data
        __syncthreads();

        // Compute sum in shared memory
        C[idx] = sA[threadIdx.x] + sB[threadIdx.x];
    }
}