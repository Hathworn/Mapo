#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    // Improve data locality by precomputing bounds
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce global memory access (if further optimization needed)
    extern __shared__ float shared[];
    float *sA = shared;
    float *sB = shared + blockDim.x;

    if (i < n) {
        // Prefetch data into shared memory
        sA[threadIdx.x] = A[i];
        sB[threadIdx.x] = B[i];
        __syncthreads();

        unsigned int k = i + offset;
        if (k < n) {
            // Access shared memory instead of global memory
            C[k] = sA[threadIdx.x] + sB[threadIdx.x];
        }
    }
}