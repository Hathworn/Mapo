#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmup(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use branchless programming for bounds checking
    if (i < N)
    {
        C[i] = A[i] + B[i]; // Direct computation inside the condition
    }
}