#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling4(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 4;

    // Unrolling 4 with conditional check
    int sum = 0;
    if (idx + 3 * blockDim.x < n)
    {
        sum = g_idata[idx] + g_idata[idx + blockDim.x] + 
              g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x];
    }
    idata[tid] = sum; // Store the sum directly

    __syncthreads();

    // In-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        // Perform reduction only if within stride
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads(); // Synchronize within threadblock
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}