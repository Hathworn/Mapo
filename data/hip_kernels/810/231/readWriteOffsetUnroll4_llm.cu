#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readWriteOffsetUnroll4(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    unsigned int k = i + offset;

    // Use loop unrolling for reduced loop overhead and better pipelining
    #pragma unroll
    for (int j = 0; j < 4; ++j)
    {
        if (k + j * blockDim.x < n)
        {
            C[k + j * blockDim.x] = A[k + j * blockDim.x] + B[k + j * blockDim.x];
        }
    }
}