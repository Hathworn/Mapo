#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Utilize a single index calculation for better memory coalescing
    unsigned int idx = blockIdx.y * blockDim.y * nx + blockIdx.x * blockDim.x + threadIdx.y * nx + threadIdx.x;

    if (idx < nx * ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}