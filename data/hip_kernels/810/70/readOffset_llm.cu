#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Use unsigned int for indexing and compute k efficiently
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Prefetch k to avoid recomputation
    unsigned int k = i + offset;

    // Process only valid elements, use fewer branches
    if (k < n) 
    {
        C[i] = A[k] + B[k];
    }
}