#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling8New(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // Unrolling 8 with boundary check
    int tmp = 0;
    if (idx + 7 * blockDim.x < n)
    {
        // Improved unrolling with a single pointer update
        for (int i = 0; i < 8; i++)
        {
            tmp += g_idata[idx + i * blockDim.x];
        }
    }
    
    // Use shared memory for reduction to enhance memory access efficiency
    __shared__ int sharedData[256];
    sharedData[tid] = tmp;
    
    __syncthreads();

    // In-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            sharedData[tid] += sharedData[tid + stride];
        }

        // Synchronize within threadblock
        __syncthreads();
    }

    // Write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sharedData[0];
}