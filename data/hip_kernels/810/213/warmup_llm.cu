#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *out, float *in, const int nx, const int ny)
{
    // Use shared memory to improve memory access efficiency
    __shared__ float tile[TILE_WIDTH][TILE_WIDTH]; 

    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        unsigned int idx = iy * nx + ix;
        tile[threadIdx.y][threadIdx.x] = in[idx]; // Load data into shared memory
        __syncthreads(); // Synchronize to ensure all threads have loaded their data

        out[idx] = tile[threadIdx.y][threadIdx.x]; // Write data from shared memory
    }
}