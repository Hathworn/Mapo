#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define IPAD 0 // Define IPAD if not defined elsewhere

__global__ void transposeSmemPadDyn(float *out, float *in, int nx, int ny)
{
    // static shared memory with padding
    extern __shared__ float tile[];

    // coordinate in original matrix
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // linear global memory index for original matrix
    unsigned int ti = iy * nx + ix;

    // thread index in transposed block
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int row_idx = threadIdx.y * (blockDim.x + IPAD) + threadIdx.x;
    unsigned int irow = idx / blockDim.y;
    unsigned int icol = idx % blockDim.y;
    unsigned int col_idx = icol * (blockDim.x + IPAD) + irow;

    // coordinate in transposed matrix
    unsigned int ix_new = blockDim.y * blockIdx.y + icol;
    unsigned int iy_new = blockDim.x * blockIdx.x + irow;

    // linear global memory index for transposed matrix
    unsigned int to = iy_new * ny + ix_new;

    // transpose with boundary test
    if (ix < nx && iy < ny)
    {
        // load data from global memory to shared memory
        tile[row_idx] = in[ti];

        // thread synchronization
        __syncthreads();

        // store data to global memory from shared memory
        // Check boundaries of the transposed index before storing
        if (ix_new < ny && iy_new < nx)
        {
            out[to] = tile[col_idx];
        }
    }
}