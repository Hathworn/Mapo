#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_add_wavelet(float *g_u2, float wavelets, const int nx, const int ny, const int ngpus)
{
    // Calculate global grid index more efficiently using constant expressions
    int ipos = (ngpus == 2) ? (ny - 10) : (ny / 2 - 10);
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idx = ipos * nx + ix;

    // Use bitwise operation for better performance with integer division by 2
    if (ix == (nx >> 1)) g_u2[idx] += wavelets;
}