#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceCompleteUnrollWarps8 (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n)
    {
        // Load values and reduce directly
        int sum = g_idata[idx] + g_idata[idx + blockDim.x] + g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x]
                + g_idata[idx + 4 * blockDim.x] + g_idata[idx + 5 * blockDim.x] + g_idata[idx + 6 * blockDim.x] + g_idata[idx + 7 * blockDim.x];
        idata[tid] = sum; // Store the reduction result in shared memory
    }
    else {
        idata[tid] = 0; // Ensure all threads initialize their positions
    }

    __syncthreads();

    // Reduce in shared memory
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();

    // Unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = idata; // Use volatile pointer for warp-level reduction
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // Write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}