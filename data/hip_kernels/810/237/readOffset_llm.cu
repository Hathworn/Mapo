#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffset(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;
    
    // Using shared memory to minimize redundant global memory accesses
    __shared__ float sharedA[256]; // Assumption: blockDim.x <= 256
    __shared__ float sharedB[256]; // Assumption: blockDim.x <= 256

    if (k < n) {
        sharedA[threadIdx.x] = A[k];
        sharedB[threadIdx.x] = B[k];
        __syncthreads(); // Ensure all threads have finished copying to shared memory

        C[i] = sharedA[threadIdx.x] + sharedB[threadIdx.x];
    }
}