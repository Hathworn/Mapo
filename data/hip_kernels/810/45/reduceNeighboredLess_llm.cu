#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighboredLess(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    // in-place reduction in global memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        // combine indexed elements
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}