#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void iKernel(float *src, float *dst)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < gridDim.x * blockDim.x)  // Ensure index is within bounds
    {
        dst[idx] = src[idx] * 2.0f;   // Efficient memory access
    }
}