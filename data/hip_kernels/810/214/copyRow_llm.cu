#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyRow(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Optimize memory access by using shared memory
    __shared__ float tile[TILE_DIM][TILE_DIM];

    if (ix < nx && iy < ny)
    {
        // Load input into shared memory
        tile[threadIdx.y][threadIdx.x] = in[iy * nx + ix];
        __syncthreads();

        // Write output from shared memory
        out[iy * nx + ix] = tile[threadIdx.y][threadIdx.x];
    }
}