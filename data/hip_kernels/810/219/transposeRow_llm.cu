#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeRow(float *out, float *in, const int nx, const int ny)
{
    // Calculate row and column indices for 2D block and grid
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check that indices are within matrix bounds
    if (ix < nx && iy < ny) {
        unsigned int in_index = iy * nx + ix;  // 1D input index
        unsigned int out_index = ix * ny + iy; // Transposed 1D output index
        out[out_index] = in[in_index];         // Transpose by writing into correct position
    }
}