#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iKernel(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory to reduce global memory access latency
    __shared__ float sharedA[256];  // Adjust size as per blockDim.x if different
    __shared__ float sharedB[256];

    if (i < N) {
        sharedA[threadIdx.x] = A[i];
        sharedB[threadIdx.x] = B[i];
        
        __syncthreads();  // Ensure all data is loaded before proceeding
        
        C[i] = sharedA[threadIdx.x] + sharedB[threadIdx.x];
    }
}