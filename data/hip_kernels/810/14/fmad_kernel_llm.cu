#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fmad_kernel(double x, double y, double *out)
{
    // Using shared memory to reduce global memory access
    __shared__ double shared_x;
    __shared__ double shared_y;
    
    if (threadIdx.x == 0)
    {
        shared_x = x;
        shared_y = y;
    }
    __syncthreads();  // Ensure shared memory is initialized
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid == 0)
    {
        *out = shared_x * shared_x + shared_y;  // Access shared memory
    }
}