#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *g_data, float value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory to reduce global memory accesses
    extern __shared__ float sdata[];
    sdata[threadIdx.x] = g_data[idx];

    __syncthreads(); // Synchronize threads within the block

    // Perform computation in shared memory
    sdata[threadIdx.x] += value;

    __syncthreads(); // Synchronize threads before writing back

    // Write back to global memory
    g_data[idx] = sdata[threadIdx.x];
}