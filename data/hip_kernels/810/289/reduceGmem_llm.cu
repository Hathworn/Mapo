#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceGmem(int *g_idata, int *g_odata, unsigned int n) {
    // Efficient set of local thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + tid;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundaries check to avoid excess calculations
    if (idx >= n) return;
    
    // Optimize in-place reduction in global memory using loop for better maintenance
    for (unsigned int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads(); // Ensure all memory operations are completed
    }
    
    // Unroll loop for warp size reduction
    if (tid < 32) {
        volatile int *vsmem = idata; // Ensure consistency within warp
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // Write results to global memory if thread 0
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}