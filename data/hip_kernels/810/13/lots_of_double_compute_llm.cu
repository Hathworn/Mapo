#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lots_of_double_compute(double *inputs, int N, size_t niters, double *outputs)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t nthreads = gridDim.x * blockDim.x;

    // Use shared memory for inputs to reduce global memory access latency
    __shared__ double shared_inputs[1024];

    for (; tid < N; tid += nthreads)
    {
        size_t iter;
        double val;

        // Load inputs into shared memory
        if (threadIdx.x < N)
            shared_inputs[threadIdx.x] = inputs[tid];

        __syncthreads();  // Ensure all threads have loaded their data

        if (threadIdx.x < N)
            val = shared_inputs[threadIdx.x];  // Access shared memory

        for (iter = 0; iter < niters; iter++)
        {
            val = (val + 5.0) - 101.0;
            val = (val / 3.0) + 102.0;
            val = (val + 1.07) - 103.0;
            val = (val / 1.037) + 104.0;
            val = (val + 3.00) - 105.0;
            val = (val / 0.22) + 106.0;
        }

        // Store results in global memory
        if (threadIdx.x < N)
            outputs[tid] = val;
    }
}