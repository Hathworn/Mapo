#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void refill_randoms(float *dRand, int N, hiprandState *states)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;
    hiprandState localState = states[tid];  // Load state once

    // Unroll loop by processing multiple elements per thread
    for (int i = tid; i < N; i += nthreads * 2)
    {
        dRand[i] = hiprand_uniform(&localState);
        if (i + nthreads < N) {  // Avoid out-of-bounds for the second element
            dRand[i + nthreads] = hiprand_uniform(&localState);
        }
    }

    states[tid] = localState;  // Save updated state
}