#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighboredGmem(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if (idx >= n) return;

    // In-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) // Use larger stride first
    {
        if (tid < stride) // Only active threads perform addition
        {
            idata[tid] += idata[tid + stride];
        }
        // Synchronize within threadblock
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}