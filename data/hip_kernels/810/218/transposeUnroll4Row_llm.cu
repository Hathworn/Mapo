#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll4Row(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Avoid repeated computation by storing common values
    unsigned int ti = iy * nx + ix;
    unsigned int to_base = ix * ny + iy;

    // Use shared memory for faster access
    __shared__ float tile[blockDim.y][blockDim.x * 4];

    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        tile[threadIdx.y][threadIdx.x] = in[ti];
        tile[threadIdx.y][threadIdx.x + blockDim.x] = in[ti + blockDim.x];
        tile[threadIdx.y][threadIdx.x + 2 * blockDim.x] = in[ti + 2 * blockDim.x];
        tile[threadIdx.y][threadIdx.x + 3 * blockDim.x] = in[ti + 3 * blockDim.x];
    }
    __syncthreads();

    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        out[to_base] = tile[threadIdx.y][threadIdx.x];
        out[to_base + ny * blockDim.x] = tile[threadIdx.y][threadIdx.x + blockDim.x];
        out[to_base + ny * 2 * blockDim.x] = tile[threadIdx.y][threadIdx.x + 2 * blockDim.x];
        out[to_base + ny * 3 * blockDim.x] = tile[threadIdx.y][threadIdx.x + 3 * blockDim.x];
    }
}