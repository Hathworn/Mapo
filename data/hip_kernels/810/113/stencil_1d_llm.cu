#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil_1d(float *in, float *out, int N)
{
    // shared memory with optimization for bank conflicts
    __shared__ float smem[BDIM + 2 * RADIUS];

    // optimized index calculation for global memory
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // process multiple elements per thread for better occupancy
    int total_threads = gridDim.x * blockDim.x;
    for (int i = idx; i < N; i += total_threads)
    {
        // index to shared memory for stencil calculation
        int sidx = threadIdx.x + RADIUS;

        // Read data from global memory into shared memory
        if (i < N) smem[sidx] = in[i];

        // read halo part to shared memory, add range check to prevent out of bounds access
        if (threadIdx.x < RADIUS)
        {
            if (i >= RADIUS) smem[sidx - RADIUS] = in[i - RADIUS];
            if (i + BDIM < N) smem[sidx + BDIM] = in[i + BDIM];
        }

        // Synchronize (ensure all the data is available)
        __syncthreads();

        // Apply the stencil
        float tmp = 0.0f;

        #pragma unroll
        for (int j = 1; j <= RADIUS; j++)
        {
            tmp += coef[j] * (smem[sidx + j] - smem[sidx - j]);
        }

        // store the result only if within bounds
        if (i < N) out[i] = tmp;

        // Synchronize to prevent race condition in smem usage
        __syncthreads();
    }
}