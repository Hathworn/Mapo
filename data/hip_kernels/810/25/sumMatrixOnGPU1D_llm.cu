#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Optimize to calculate both row and column indices using 2D grid
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Check boundaries for both x and y
    if (ix < nx && iy < ny) {
        int idx = iy * nx + ix;
        MatC[idx] = MatA[idx] + MatB[idx]; // Perform element-wise addition
    }
}