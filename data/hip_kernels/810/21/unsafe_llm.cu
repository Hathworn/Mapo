#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unsafe(int *shared_var, int *values_read, int N, int iters)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= N) return;

    // Using atomicAdd to avoid race conditions
    int old = atomicAdd(shared_var, 1);
    values_read[tid] = old;

    for (int i = 0; i < iters; i++)
    {
        atomicAdd(shared_var, 1);
    }
}