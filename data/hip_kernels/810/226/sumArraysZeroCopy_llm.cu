#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysZeroCopy(float *A, float *B, float *C, const int N)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check for boundary condition and perform addition
    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}