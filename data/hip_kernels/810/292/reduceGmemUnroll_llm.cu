#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceGmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    
    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 4;

    // Unrolling 4
    int mySum = 0;
    if (idx + 3 * blockDim.x < n) {
        mySum = g_idata[idx] + g_idata[idx + blockDim.x] + 
                g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x];
    }
    
    // Store the sum in shared memory
    idata[tid] = mySum;
    __syncthreads();

    // In-place reduction in global memory
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) idata[tid] += idata[tid + s];
        __syncthreads();
    }

    // Unrolling warp
    if (tid < 32) {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}