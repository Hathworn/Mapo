#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1d(float *in, float *out)
{
    // shared memory
    __shared__ float smem[BDIM + 2 * RADIUS];

    // index to global memory
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // index to shared memory for stencil calculatioin
    int sidx = threadIdx.x + RADIUS;

    // Efficient data coalescing into shared memory
    smem[sidx] = in[idx];
    if (threadIdx.x < RADIUS) {
        smem[sidx - RADIUS] = in[idx - RADIUS];
        smem[sidx + BDIM] = in[idx + BDIM];
    }

    // Synchronize (ensure all the data is available)
    __syncthreads();

    // Apply the stencil with loop unrolling
    float tmp = 0.0f;
    #pragma unroll
    for (int i = 1; i <= RADIUS; i++) {
        tmp += coef[i] * (smem[sidx + i] - smem[sidx - i]);
    }

    // Store the result
    out[idx] = tmp;
}