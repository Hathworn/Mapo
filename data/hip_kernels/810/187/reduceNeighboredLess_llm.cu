#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighboredLess (int *g_idata, int *g_odata, unsigned int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    // using shared memory for faster access
    extern __shared__ int sdata[];
    sdata[tid] = idata[tid];
    __syncthreads();

    // optimized in-place reduction with fewer synchronizations
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}