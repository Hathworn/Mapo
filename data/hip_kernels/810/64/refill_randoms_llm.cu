#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refill_randoms(float *dRand, int N, hiprandState *states)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Compute the stride for the grid

    hiprandState localState = states[tid]; // Use local copy of the state

    for (int i = tid; i < N; i += stride) // Use stride instead of nthreads
    {
        dRand[i] = hiprand_uniform(&localState); // Use local state for random generation
    }
    
    states[tid] = localState; // Save the state back
}