#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate 1D global index for current thread
    unsigned int idx = blockIdx.y * blockDim.y * nx + blockIdx.x * blockDim.x + threadIdx.y * nx + threadIdx.x;

    // Boundary check
    if (idx < nx * ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx]; // Perform element-wise addition
    }
}