#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceSmemUnrollDyn(int *g_idata, int *g_odata, unsigned int n) {
    extern __shared__ int smem[];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // unrolling 4
    int tmpSum = (idx + 3 * blockDim.x < n) ? g_idata[idx] + g_idata[idx + blockDim.x] + g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x] : 0;

    smem[tid] = tmpSum;
    __syncthreads();

    // in-place reduction in shared memory
    if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];
    __syncthreads();

    // unrolling warp
    if (tid < 32) {
        // Use volatile pointer to ensure compiler doesn't optimize away memory accesses
        volatile int *vsmem = smem;
        // Warp synchronous reduction using loop
        for (int i = 32; i > 0; i >>= 1) {
            vsmem[tid] += vsmem[tid + i];
        }
    }

    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}