#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void naiveGmemUnroll(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned int ti = iy * nx + ix;
    unsigned int to = ix * ny + iy;

    if (ix < nx && iy < ny) // Ensure threads within bounds
    {
        out[to] = in[ti];
        if (ix + blockDim.x < nx) // Second read only if within bounds
        {
            out[to + ny * blockDim.x] = in[ti + blockDim.x];
        }
    }
}