#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmemDyn(int *g_idata, int *g_odata, unsigned int n)
{
    extern __shared__ int smem[];

    // Set thread ID
    unsigned int tid = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Load data to shared memory
    smem[tid] = idata[tid];
    __syncthreads();

    // In-place reduction using shared memory
    for (int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            smem[tid] += smem[tid + s];
        }
        __syncthreads();
    }

    // Unrolled loop for last warp
    if (tid < 32) {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}