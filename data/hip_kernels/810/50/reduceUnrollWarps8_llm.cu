#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrollWarps8(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    int sum = 0;
    if (idx + 7 * blockDim.x < n)
    {
        sum += g_idata[idx];
        sum += g_idata[idx + blockDim.x];
        sum += g_idata[idx + 2 * blockDim.x];
        sum += g_idata[idx + 3 * blockDim.x];
        sum += g_idata[idx + 4 * blockDim.x];
        sum += g_idata[idx + 5 * blockDim.x];
        sum += g_idata[idx + 6 * blockDim.x];
        sum += g_idata[idx + 7 * blockDim.x];
    }
    idata[tid] = sum; // Store the local sum

    __syncthreads();

    // in-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        // synchronize within threadblock
        __syncthreads();
    }

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}