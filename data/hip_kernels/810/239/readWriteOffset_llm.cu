#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readWriteOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate once to improve efficiency
    unsigned int k = blockIdx.x * blockDim.x + threadIdx.x + offset;

    // Ensure calculations only occur if within bounds
    if (k < n)
    {
        C[k] = A[k] + B[k];
    }
}