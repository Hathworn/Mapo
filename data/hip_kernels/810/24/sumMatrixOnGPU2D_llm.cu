#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Use 2D block and grid dimensions to optimize thread indexing
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < nx && iy < ny) // Ensure indices are within matrix dimensions
    {
        unsigned int idx = iy * nx + ix; // Efficient shared index calculation
        MatC[idx] = MatA[idx] + MatB[idx]; // Perform the addition operation
    }
}