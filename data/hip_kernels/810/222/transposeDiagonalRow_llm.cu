#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeDiagonalRow(float *out, float *in, const int nx, const int ny)
{
    // Optimize block and thread index calculation
    unsigned int ix = blockIdx.y * blockDim.x + threadIdx.x;  // Use blockIdx.y for row-wise block distribution
    unsigned int iy = blockIdx.x * blockDim.y + threadIdx.y;  // Use blockIdx.x for column-wise block distribution

    if (ix < nx && iy < ny)
    {
        // Perform transposed memory access
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}