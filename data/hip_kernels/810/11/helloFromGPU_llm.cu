#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU()
{
    // Launch the kernel with a single thread.
    // This avoids unnecessary blocks and threads.
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        printf("Hello World from GPU!\n");
    }
}