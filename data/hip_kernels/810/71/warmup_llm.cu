#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *out, float *in, const int nx, const int ny)
{
    // Calculate global thread index
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Linearize 2D block index fetching
    unsigned int idx = iy * nx + ix;

    // Only compute valid indices
    if (ix < nx && iy < ny)
    {
        out[idx] = in[idx];
    }
}