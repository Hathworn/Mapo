#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lots_of_float_compute(float *inputs, int N, size_t niters, float *outputs)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t nthreads = gridDim.x * blockDim.x;

    // Loop unrolling to reduce loop overhead and potentially increase instruction level parallelism
    for ( ; tid < N; tid += nthreads)
    {
        float val = inputs[tid];

        for (size_t iter = 0; iter < niters; iter += 4) // step by 4 and handle each loop separately
        {
            val = (val + 5.0f) - 101.0f;
            val = (val / 3.0f) + 102.0f;
            val = (val + 1.07f) - 103.0f;
            val = (val / 1.037f) + 104.0f;
            val = (val + 3.00f) - 105.0f;
            val = (val / 0.22f) + 106.0f;
                        
            val = (val + 5.0f) - 101.0f;
            val = (val / 3.0f) + 102.0f;
            val = (val + 1.07f) - 103.0f;
            val = (val / 1.037f) + 104.0f;
            val = (val + 3.00f) - 105.0f;
            val = (val / 0.22f) + 106.0f;
                        
            val = (val + 5.0f) - 101.0f;
            val = (val / 3.0f) + 102.0f;
            val = (val + 1.07f) - 103.0f;
            val = (val / 1.037f) + 104.0f;
            val = (val + 3.00f) - 105.0f;
            val = (val / 0.22f) + 106.0f;
                        
            val = (val + 5.0f) - 101.0f;
            val = (val / 3.0f) + 102.0f;
            val = (val + 1.07f) - 103.0f;
            val = (val / 1.037f) + 104.0f;
            val = (val + 3.00f) - 105.0f;
            val = (val / 0.22f) + 106.0f;
        }

        outputs[tid] = val;
    }
}