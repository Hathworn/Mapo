#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNaiveCol(float *out, float *in, const int nx, const int ny)
{
    // Compute combined column-major index
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Perform bounds check only after computing indices
    if (ix < nx && iy < ny)
    {
        // Coalesced memory access pattern by writing in row-major order
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}