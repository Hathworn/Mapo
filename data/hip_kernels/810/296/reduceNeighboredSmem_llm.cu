#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighboredSmem(int *g_idata, int *g_odata, unsigned int n)
{
    __shared__ int smem[DIM];

    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (idx >= n) return;

    // Load the global data into shared memory
    smem[tid] = g_idata[idx];
    __syncthreads();

    // In-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) // Start from blockDim.x / 2 and halve stride each iteration
    {
        if (tid < stride) // Only threads whose IDs are less than stride participate
        {
            smem[tid] += smem[tid + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}