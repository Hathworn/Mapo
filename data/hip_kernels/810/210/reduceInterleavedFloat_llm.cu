#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceInterleavedFloat(float *g_idata, float *g_odata, unsigned int n)
{
    // Set thread ID and calculate global index
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    float* idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if (idx >= n) return;

    // In-place reduction using shared memory for optimization
    extern __shared__ float sdata[];
    sdata[tid] = idata[tid];
    __syncthreads();

    // Perform reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}