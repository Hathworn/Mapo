#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(int *MatA, int *MatB, int *MatC, int nx, int ny)
{
    // Calculate global index more efficiently using offsets
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;

    // Ensure within bounds using conditional execution
    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}