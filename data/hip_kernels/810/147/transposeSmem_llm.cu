#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeSmem(float *out, float *in, int nx, int ny)
{
    // Static shared memory with padding to avoid bank conflicts
    __shared__ float tile[BDIMY][BDIMX + 1];

    // Coordinate in original matrix
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Linear global memory index for original matrix
    unsigned int ti = iy * nx + ix;

    // Thread index in transposed block
    unsigned int bidx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = bidx / blockDim.y;
    unsigned int icol = bidx % blockDim.y;

    // Coordinate in transposed matrix
    unsigned int trans_ix = blockDim.y * blockIdx.y + icol;
    unsigned int trans_iy = blockDim.x * blockIdx.x + irow;

    // Linear global memory index for transposed matrix
    unsigned int to = trans_iy * ny + trans_ix;

    // Transpose with boundary check
    if (ix < nx && iy < ny)
    {
        // Load data from global memory to shared memory
        tile[threadIdx.y][threadIdx.x] = in[ti];
    }

    // Thread synchronization
    __syncthreads();

    // Store data to global memory from shared memory
    if (trans_ix < ny && trans_iy < nx)
    {
        out[to] = tile[icol][irow];
    }
}