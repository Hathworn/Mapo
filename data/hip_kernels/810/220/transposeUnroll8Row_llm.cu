#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll8Row(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 8 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    // Ensure enough threads are launched to cover the rows in x
    if (ix < nx && iy < ny)
    {
        #pragma unroll
        for (int i = 0; i < 8 && ix + i * blockDim.x < nx; ++i)
        {
            out[to + i * ny * blockDim.x] = in[ti + i * blockDim.x];
        }
    }
}