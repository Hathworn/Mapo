#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Use shared memory for faster access
    __shared__ float sharedA[BLOCK_SIZE_Y][BLOCK_SIZE_X];
    __shared__ float sharedB[BLOCK_SIZE_Y][BLOCK_SIZE_X];

    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    // Load data into shared memory
    if (ix < nx && iy < ny) {
        sharedA[threadIdx.y][threadIdx.x] = MatA[idx];
        sharedB[threadIdx.y][threadIdx.x] = MatB[idx];
    }
    __syncthreads(); // Synchronize to ensure all threads have loaded their data

    // Compute result using shared memory
    if (ix < nx && iy < ny)
        MatC[idx] = sharedA[threadIdx.y][threadIdx.x] + sharedB[threadIdx.y][threadIdx.x];
}
```
