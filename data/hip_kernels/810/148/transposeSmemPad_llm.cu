#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeSmemPad(float *out, float *in, int nx, int ny)
{
    // Use static shared memory with padding
    __shared__ float tile[BDIMY][BDIMX + IPAD];

    // Get x and y coordinates in original matrix
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Calculate global memory index for original matrix
    unsigned int ti = iy * nx + ix;

    // Transpose matrix indices using optimized thread calculation
    // Determine thread index within the transposed block
    unsigned int irow = threadIdx.x;
    unsigned int icol = threadIdx.y;

    // Calculate coordinates in transposed matrix
    unsigned int tix = blockDim.y * blockIdx.y + icol;
    unsigned int tiy = blockDim.x * blockIdx.x + irow;

    // Calculate linear global memory index for transposed matrix
    unsigned int to = tiy * ny + tix;

    // Only proceed if within matrix bounds
    if (ix < nx && iy < ny)
    {
        // Load data from global memory to shared memory
        tile[icol][irow] = in[ti];

        // Synchronize threads for consistent shared memory access
        __syncthreads();

        // Store transposed data to global memory from shared memory
        out[to] = tile[threadIdx.y][threadIdx.x];
    }
}