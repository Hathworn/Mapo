#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeDiagonalCol(float *out, float *in, const int nx, const int ny)
{
    // Optimize block and grid setup for better memory coalescing and access patterns
    unsigned int blk_y = blockIdx.y;
    unsigned int blk_x = blockIdx.x;

    // Swap block dimension indexing to spread workload effectively
    unsigned int ix = blockDim.y * blk_x + threadIdx.y;
    unsigned int iy = blockDim.x * blk_y + threadIdx.x;

    // Check the bounds for the transposed indices
    if (ix < ny && iy < nx)
    {
        out[ix * nx + iy] = in[iy * ny + ix];
    }
}