#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrollWarps(int *g_idata, int *g_odata, unsigned int n) {
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // Unrolling 2
    if (idx + blockDim.x < n) g_idata[idx] += g_idata[idx + blockDim.x];

    __syncthreads();

    // In-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }

        // Synchronize within threadblock
        __syncthreads();
    }

    // Unrolling last warp using volatile
    if (tid < 32) {
        volatile int *vsmem = idata;
        
        // Use loop to reduce code repetition
        #pragma unroll
        for (int stride = 32; stride >= 1; stride /= 2) {
            vsmem[tid] += vsmem[tid + stride];
        }
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}