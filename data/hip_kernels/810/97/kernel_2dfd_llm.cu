#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_2dfd(float *g_u1, float *g_u2, const int nx, const int iStart, const int iEnd)
{
    // global to line index
    unsigned int ix  = blockIdx.x * blockDim.x + threadIdx.x;

    // smem idx for current point
    unsigned int stx = threadIdx.x + NPAD;
    unsigned int idx  = ix + iStart * nx;

    // shared memory for x dimension
    __shared__ float line[BDIMX + NPAD2];

    // a coefficient related to physical properties
    const float alpha = 0.12f;

    // register for y value
    float yval[9];

    // Prefetch the initial values to reduce latency
    #pragma unroll
    for (int i = 0; i < 8; i++) yval[i] = g_u2[idx + (i - 4) * nx];

    // skipping for the bottom most y value
    int iskip = NPAD * nx;

    #pragma unroll 9
    for (int iy = iStart; iy < iEnd; iy++)
    {
        // get yval[8] here
        yval[8] = g_u2[idx + iskip];

        // read halo part and avoid bank conflicts by padding
        if(threadIdx.x < NPAD)
        {
            line[threadIdx.x]  = g_u2[idx - NPAD];
            line[stx + BDIMX] = g_u2[idx + BDIMX];
        }

        line[stx] = yval[4];
        __syncthreads();

        // 8th order finite-difference operator
        if ( (ix >= NPAD) && (ix < nx - NPAD) )
        {
            // center point
            float tmp = coef[0] * line[stx] * 2.0f;

            #pragma unroll
            for(int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (line[stx - d] + line[stx + d]);
            }

            #pragma unroll
            for(int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (yval[4 - d] + yval[4 + d]);
            }

            // time dimension
            g_u1[idx] = yval[4] + yval[4] - g_u1[idx] + alpha * tmp;
        }

        // Shift yval to the left and bring in new value
        #pragma unroll
        for (int i = 0; i < 8 ; i++)
        {
            yval[i] = yval[i + 1];
        }

        // advance on global idx
        idx  += nx;
        __syncthreads();
    }
}