#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void host_api_kernel(float *randomValues, float *out, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop through the range of indices this thread is responsible for, stride by nthreads
    for (int i = tid; i < N; i += gridDim.x * blockDim.x)
    {
        // Directly store the result to the output array
        out[i] = randomValues[i] * 2;
    }
}