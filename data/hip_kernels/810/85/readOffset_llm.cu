#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate the global index only once
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Use grid-stride loop to handle larger data
    for (; k < n; k += blockDim.x * gridDim.x)
    {
        C[k - offset] = A[k] + B[k];
    }
}