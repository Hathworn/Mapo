#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, int NX, int NY)
{
    // Use 2D grid and block indices for better readability and clarity
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Optimize boundary check using a single condition
    if (ix < NX && iy < NY) 
    {
        unsigned int idx = iy * NX + ix; // Calculate index only when within bounds
        C[idx] = A[idx] + B[idx];
    }
}