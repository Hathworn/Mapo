#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmem(int *g_idata, int *g_odata, unsigned int n)
{
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n) return;

    // load data to shared memory
    smem[tid] = g_idata[idx];
    __syncthreads();

    // unrolled loop for in-place reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            smem[tid] += smem[tid + s];
        }
        __syncthreads(); // ensure all adds complete before next stage
    }

    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}