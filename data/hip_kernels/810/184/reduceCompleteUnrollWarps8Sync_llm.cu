#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceCompleteUnrollWarps8Sync(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // Unrolling 8
    int sum = 0;
    if (idx + 7 * blockDim.x < n) {
        for (int i = 0; i < 8; i++) {
            sum += g_idata[idx + i * blockDim.x];
        }
        g_idata[idx] = sum;
    }

    __syncthreads();

    // In-place reduction using loop to replace repeated code
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            idata[tid] += idata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}