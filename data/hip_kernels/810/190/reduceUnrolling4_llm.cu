#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling4 (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 4;

    // use register to store values for unrolling
    int sum = 0;

    // unrolling 4
    if (idx + 3 * blockDim.x < n)
    {
        sum += g_idata[idx];
        sum += g_idata[idx + blockDim.x];
        sum += g_idata[idx + 2 * blockDim.x];
        sum += g_idata[idx + 3 * blockDim.x];
        
        // store the accumulated sum back to the original position
        idata[tid] = sum;
    }
    else
    {
        if (idx < n) sum += g_idata[idx];
        if (idx + blockDim.x < n) sum += g_idata[idx + blockDim.x];
        if (idx + 2 * blockDim.x < n) sum += g_idata[idx + 2 * blockDim.x];
        if (idx + 3 * blockDim.x < n) sum += g_idata[idx + 3 * blockDim.x];
        
        idata[tid] = sum;
    }

    __syncthreads();

    // in-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads(); // synchronize within threadblock
    }

    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}