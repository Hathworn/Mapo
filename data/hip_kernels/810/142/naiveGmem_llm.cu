#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void naiveGmem(float *out, float *in, const int nx, const int ny)
{
    // Calculate global thread index for the output and input
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Precompute linear index for better memory coalescing
    unsigned int outIdx = ix * ny + iy;
    unsigned int inIdx = iy * nx + ix;

    // Transpose with boundary check
    if (ix < nx && iy < ny)
    {
        out[outIdx] = in[inIdx]; // Direct array index usage
    }
}