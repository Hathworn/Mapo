#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeDiagonalColUnroll4(float *out, float *in, const int nx, const int ny)
{
    // Use shared memory to improve memory access speed
    __shared__ float tile[4][4];

    unsigned int blk_y = blockIdx.x;
    unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;
    
    unsigned int ix_stride = blockDim.x * blk_x;
    unsigned int ix = ix_stride * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;
    
    if (ix < nx && iy < ny)
    {
        tile[threadIdx.y][threadIdx.x] = in[ix * ny + iy];
        __syncthreads();

        if (threadIdx.y < 4) {
            out[iy * nx + ix] = tile[threadIdx.y][threadIdx.x];
            ix += blockDim.x;
            out[iy * nx + ix] = tile[threadIdx.y][threadIdx.x];
            ix += blockDim.x;
            out[iy * nx + ix] = tile[threadIdx.y][threadIdx.x];
            ix += blockDim.x;
            out[iy * nx + ix] = tile[threadIdx.y][threadIdx.x];
        }
    }
}