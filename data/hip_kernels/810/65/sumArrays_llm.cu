#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll the loop to improve memory access
    #pragma unroll
    for (; i < N; i += blockDim.x * gridDim.x) {
        C[i] = A[i] + B[i];
    }
}