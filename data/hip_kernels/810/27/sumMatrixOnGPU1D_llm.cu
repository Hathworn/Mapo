#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate global thread ID
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate global index and stride
    unsigned int idx = iy * nx + ix;
    unsigned int stride = blockDim.x * gridDim.x;

    // Ensure thread is within bounds and process in strides
    for (unsigned int i = idx; i < nx * ny; i += stride)
    {
        MatC[i] = MatA[i] + MatB[i];
    }
}