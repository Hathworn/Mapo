#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_add_wavelet(float *g_u2, float wavelets, const int nx, const int ny, const int ngpus)
{
    // Calculate global index on (x, y) plane
    int ipos = (ngpus == 2 ? ny - 10 : ny / 2 - 10);
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread is within bounds
    if (ix < nx) {
        unsigned int idx = ipos * nx + ix;
        
        // Use atomic operation for thread safety
        if (ix == nx / 2) {
            atomicAdd(&g_u2[idx], wavelets);
        }
    }
}