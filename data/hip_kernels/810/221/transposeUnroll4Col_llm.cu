#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeUnroll4Col(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Calculate shared memory index for coalesced access
    unsigned int ti = iy * nx + ix;
    unsigned int to = ix * ny + iy;

    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        // Use registers for repeated calculations
        unsigned int inputIndex1 = to;
        unsigned int inputIndex2 = to + blockDim.x * ny;
        unsigned int inputIndex3 = to + 2 * blockDim.x * ny;
        unsigned int inputIndex4 = to + 3 * blockDim.x * ny;

        // Transfer data with less global memory access
        float temp1 = in[inputIndex1];
        float temp2 = in[inputIndex2];
        float temp3 = in[inputIndex3];
        float temp4 = in[inputIndex4];

        // Writing back to global memory
        out[ti] = temp1;
        out[ti +   blockDim.x] = temp2;
        out[ti + 2 * blockDim.x] = temp3;
        out[ti + 3 * blockDim.x] = temp4;
    }
}