#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNaiveCol(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Efficient memory access by converting 2D index to 1D
    if (ix < nx && iy < ny)
    {
        unsigned int in_index = ix * ny + iy;
        unsigned int out_index = iy * nx + ix;
        out[out_index] = in[in_index];
    }
}