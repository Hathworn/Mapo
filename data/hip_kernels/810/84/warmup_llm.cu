#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Use a check to ensure valid memory access
    if (i < n) 
    {
        // Compute only if the index is valid
        C[i] = A[k] + B[k];
    }
}