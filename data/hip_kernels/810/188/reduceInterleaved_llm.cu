#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceInterleaved(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if (idx >= n) return;

    // In-place reduction in shared memory for faster access
    extern __shared__ int sdata[];
    sdata[tid] = idata[tid];
    __syncthreads();

    // Reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            // Perform reduction in shared memory
            sdata[tid] += sdata[tid + stride];
        }

        // Synchronize threads at each reduction step
        __syncthreads();
    }

    // Write result for this block to global memory from shared memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}