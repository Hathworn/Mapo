#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeDiagonalRow(float *out, float *in, const int nx, const int ny)
{
    // Cache grid dimensions
    unsigned int gridDimX = gridDim.x;
    unsigned int gridDimY = gridDim.y;

    // Improved block indexing for better coalescing and access patterns
    unsigned int blk_x = blockIdx.x;
    unsigned int blk_y = (blockIdx.x + blockIdx.y) % gridDimY;

    // Calculate thread's output matrix row and column
    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    // Check bounds and transpose
    if (ix < nx && iy < ny)
    {
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}