#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_reduction(int *shared_var, int *input_values, int N, int iters)
{
    __shared__ int local_mem[256];
    int iter;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    for (iter = 0; iter < iters; iter++)
    {
        if (tid < N)
        {
            local_mem[local_tid] = input_values[tid];
        }

        // Synchronize to make sure all threads have loaded their data
        __syncthreads();

        // Perform binary tree reduction
        for (int stride = 1; stride < blockDim.x; stride *= 2)
        {
            if (local_tid % (2 * stride) == 0 && (local_tid + stride < blockDim.x))
            {
                local_mem[local_tid] += local_mem[local_tid + stride];
            }

            // Synchronize after each reduction step to ensure all operations are complete
            __syncthreads();
        }

        // Final update to shared_var using atomic operation
        if (local_tid == 0)
        {
            atomicAdd(shared_var, local_mem[0]);
        }
    }
}