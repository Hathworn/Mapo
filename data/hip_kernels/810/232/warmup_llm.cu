#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Ensure k is within bounds and utilize shared memory for faster access
    if (k < n) {
        extern __shared__ float sdata[];
        sdata[threadIdx.x] = A[i] + B[i]; // Load to shared memory
        __syncthreads(); // Ensure all loads to shared memory are complete
        C[k] = sdata[threadIdx.x]; // Store result from shared memory to global memory
    }
}