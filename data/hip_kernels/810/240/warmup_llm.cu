#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;
    
    // Use memory coalescing by ensuring data access patterns are aligned
    if (k < n) 
    {
        C[k] = A[k - offset] + B[k - offset];
    }
}