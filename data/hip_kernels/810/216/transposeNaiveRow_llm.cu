#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNaiveRow(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        // Optimize memory access by loading a tile into shared memory
        __shared__ float tile[32][32];

        unsigned int tidx = threadIdx.x;
        unsigned int tidy = threadIdx.y;

        // Load data into shared memory
        tile[tidy][tidx] = in[iy * nx + ix];
        __syncthreads();

        // Transpose and write back to global memory
        out[ix * ny + iy] = tile[tidx][tidy];
    }
}