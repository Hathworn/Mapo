#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void intrinsic_kernel(float a, float *out, int iters)
{
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

    if(tid == 0)
    {
        // Reduce register usage by moving tmp declaration outside the loop
        float tmp = __powf(a, 2.0f);

        // Remove unnecessary repetitive computation in loop
        for (int i = 1; i < iters; i++) { }

        *out = tmp;
    }
}