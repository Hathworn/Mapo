#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1d_global(float* in, float* out, float* dcoef) {
    // shared memory
    __shared__ float smem[BDIM + 2 * RADIUS];

    // index to global memory
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // index to shared memory for stencil calculations
    int sidx = threadIdx.x + RADIUS;

    // Ensure the index is within bounds
    if (idx < RADIUS || idx >= BDIM - RADIUS) return;

    // Read data from global memory into shared memory
    smem[sidx] = in[idx];

    // read halo part to shared memory
    if (threadIdx.x < RADIUS) {
        smem[sidx - RADIUS] = in[idx - RADIUS];
        smem[sidx + BDIM] = in[idx + BDIM];
    }

    // Synchronize (ensure all the data is available)
    __syncthreads();

    // Apply the stencil using unrolling
    float tmp = 0.0f;
    #pragma unroll
    for (int i = 1; i <= RADIUS; i++) {
        tmp += dcoef[i] * (smem[sidx + i] - smem[sidx - i]);
    }

    // Store the result
    out[idx] = tmp;
}