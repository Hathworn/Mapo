#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceGmem(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    // in-place reduction in global memory
    for (unsigned int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }
    
    // unrolling warp for the last part
    if (tid < 32) {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}