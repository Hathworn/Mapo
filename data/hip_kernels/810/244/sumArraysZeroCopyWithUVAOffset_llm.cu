#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysZeroCopyWithUVAOffset(float *A, float *B, float *C, const int N, int offset)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to cache offset value for faster access
    __shared__ int sharedOffset;
    if (threadIdx.x == 0) {
        sharedOffset = offset;
    }
    __syncthreads();

    // Perform calculation only for valid indices
    if (i < N) {
        int idx = i + sharedOffset;  // Use cached offset
        C[idx] = A[idx] + B[idx];
    }
}