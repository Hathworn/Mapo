#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a more cache-friendly data access pattern
    if (i < N) {
        float valA = A[i];
        float valB = B[i];
        C[i] = valA + valB;
    }
}