#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrollWarps(int *g_idata, int *g_odata, unsigned int n) {
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // Unrolling 2
    if (idx + blockDim.x < n) {
        g_idata[idx] += g_idata[idx + blockDim.x];
    }
    __syncthreads();

    // In-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        // synchronize within threadblock
        __syncthreads();
    }

    // Unrolling last warp
    if (tid < 32) {
        volatile int *vsmem = idata;
        // Unroll the loop to avoid branch divergence
        if (blockDim.x >= 64) vsmem[tid] += vsmem[tid + 32];
        if (blockDim.x >= 32) vsmem[tid] += vsmem[tid + 16];
        if (blockDim.x >= 16) vsmem[tid] += vsmem[tid + 8];
        if (blockDim.x >= 8) vsmem[tid] += vsmem[tid + 4];
        if (blockDim.x >= 4) vsmem[tid] += vsmem[tid + 2];
        if (blockDim.x >= 2) vsmem[tid] += vsmem[tid + 1];
    }

    // Output the result for this block
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}