#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling2 (int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // Unrolling 2: Load data into shared memory if within boundary
    if (idx < n) {
        idata[tid] = g_idata[idx];
        if (idx + blockDim.x < n) {
            idata[tid] += g_idata[idx + blockDim.x];
        }
    } else {
        idata[tid] = 0;
    }

    __syncthreads();

    // In-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }

        // Synchronize within threadblock
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}