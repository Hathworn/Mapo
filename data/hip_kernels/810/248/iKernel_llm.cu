#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iKernel(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop for improved performance
    if (i < N) {
        C[i] = A[i] + B[i];
    }
    i += blockDim.x * gridDim.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}