#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffsetUnroll4(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate the global index for the thread using unroll factor
    unsigned int i = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    unsigned int k = i + offset;

    // Use shared memory for coalesced access
    __shared__ float sharedA[4 * blockDim.x];
    __shared__ float sharedB[4 * blockDim.x];

    if (k + 3 * blockDim.x < n)
    {
        // Load data into shared memory
        sharedA[threadIdx.x]                 = A[k];
        sharedA[threadIdx.x + blockDim.x]    = A[k + blockDim.x];
        sharedA[threadIdx.x + 2 * blockDim.x] = A[k + 2 * blockDim.x];
        sharedA[threadIdx.x + 3 * blockDim.x] = A[k + 3 * blockDim.x];

        sharedB[threadIdx.x]                 = B[k];
        sharedB[threadIdx.x + blockDim.x]    = B[k + blockDim.x];
        sharedB[threadIdx.x + 2 * blockDim.x] = B[k + 2 * blockDim.x];
        sharedB[threadIdx.x + 3 * blockDim.x] = B[k + 3 * blockDim.x];
        
        __syncthreads(); // Synchronize here to ensure data is loaded

        // Perform computation using shared memory
        C[i]                  = sharedA[threadIdx.x]                  + sharedB[threadIdx.x];
        C[i + blockDim.x]     = sharedA[threadIdx.x + blockDim.x]     + sharedB[threadIdx.x + blockDim.x];
        C[i + 2 * blockDim.x] = sharedA[threadIdx.x + 2 * blockDim.x] + sharedB[threadIdx.x + 2 * blockDim.x];
        C[i + 3 * blockDim.x] = sharedA[threadIdx.x + 3 * blockDim.x] + sharedB[threadIdx.x + 3 * blockDim.x];
    }
}