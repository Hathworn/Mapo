#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void atomics(int *shared_var, int *values_read, int N, int iters)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= N) return;

    // Use atomicAdd once to read the value and increment
    values_read[tid] = atomicAdd(shared_var, iters + 1) + 1;
}