#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeOffsetUnroll4(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate the starting index for the current thread
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;
    
    // Use shared memory to reduce global memory latency
    __shared__ float shared_A[1024]; // Assuming maximum block size is 1024
    __shared__ float shared_B[1024];

    // Loading data into shared memory
    if (i < n) {
        shared_A[threadIdx.x] = A[i];
        shared_B[threadIdx.x] = B[i];
    }
    __syncthreads();

    // Perform operations if within the bounds
    if (k + 3 * blockDim.x < n)
    {
        // Use shared memory for faster access
        C[k]              = shared_A[threadIdx.x]               + shared_B[threadIdx.x];
        C[k + blockDim.x]   = shared_A[threadIdx.x + blockDim.x]   + shared_B[threadIdx.x + blockDim.x];
        C[k + 2 * blockDim.x] = shared_A[threadIdx.x + 2 * blockDim.x] + shared_B[threadIdx.x + 2 * blockDim.x];
        C[k + 3 * blockDim.x] = shared_A[threadIdx.x + 3 * blockDim.x] + shared_B[threadIdx.x + 3 * blockDim.x];
    }
}