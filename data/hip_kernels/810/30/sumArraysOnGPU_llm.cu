#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // Use blockIdx and blockDim for scalable parallelism
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check against N for boundary condition
    if (i < N) 
        C[i] = A[i] + B[i];
}