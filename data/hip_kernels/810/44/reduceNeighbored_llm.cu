#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighbored (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n) return;

    // in-place reduction in shared memory for faster access
    extern __shared__ int smem[];
    smem[tid] = g_idata[idx];
    __syncthreads();

    // efficient reduction using shared memory
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if (tid % (2 * stride) == 0)
        {
            smem[tid] += smem[tid + stride];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}