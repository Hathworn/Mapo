#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_api_kernel(hiprandState *states, float *out, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit if the index is out of bounds
    if (tid >= N) return;

    hiprandState localState;
    hiprand_init(9384, tid, 0, &localState);

    // Loop over elements with a stride of total number of threads
    for (int i = tid; i < N; i += blockDim.x * gridDim.x)
    {
        float rand = hiprand_uniform(&localState);
        rand = rand * 2;
        out[i] = rand;
    }
}