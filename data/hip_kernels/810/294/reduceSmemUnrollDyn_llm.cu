#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceSmemUnrollDyn(int *g_idata, int *g_odata, unsigned int n)
{
    extern __shared__ int smem[];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // unrolling 4 with bounds check
    int tmpSum = 0;
    if (idx < n) tmpSum = g_idata[idx];
    if (idx + blockDim.x < n) tmpSum += g_idata[idx + blockDim.x];
    if (idx + 2 * blockDim.x < n) tmpSum += g_idata[idx + 2 * blockDim.x];
    if (idx + 3 * blockDim.x < n) tmpSum += g_idata[idx + 3 * blockDim.x];

    smem[tid] = tmpSum;
    __syncthreads();

    // in-place reduction in global memory using a loop for more readability
    for(unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) smem[tid] += smem[tid + s];
        __syncthreads();
    }

    // unrolling the last warp
    if (tid < 32) {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}