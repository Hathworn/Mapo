#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_2dfd(float *g_u1, float *g_u2, const int nx, const int iStart, const int iEnd)
{
    // Calculate the global index
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;

    // Shared memory index for current point
    unsigned int stx = threadIdx.x + NPAD;
    unsigned int idx = ix + iStart * nx;

    // Shared memory for x dimension
    __shared__ float line[BDIMX + NPAD2];

    // A coefficient related to physical properties
    const float alpha = 0.12f;

    // Register array for y values
    float yval[9];

    // Prefetch initial data
    for (int i = 0; i < 8; i++) yval[i] = g_u2[idx + (i - 4) * nx];

    // Skip index for the bottommost y value
    int iskip = NPAD * nx;

    // Loop over the y-dimension
    #pragma unroll 9
    for (int iy = iStart; iy < iEnd; iy++)
    {
        // Fetch the next yval
        yval[8] = g_u2[idx + iskip];

        // Update shared memory with halo and current row
        if(threadIdx.x < NPAD)
        {
            line[threadIdx.x] = g_u2[idx - NPAD];
            line[stx + BDIMX] = g_u2[idx + BDIMX];
        }
        line[stx] = yval[4];
        __syncthreads();

        // 8-point finite difference operator
        if (ix >= NPAD && ix < nx - NPAD)
        {
            // Calculate center point contribution
            float tmp = coef[0] * line[stx] * 2.0f;

            // Calculate spatial contributions
            #pragma unroll
            for(int d = 1; d <= 4; d++) 
            {
                tmp += coef[d] * (line[stx - d] + line[stx + d]);
                tmp += coef[d] * (yval[4 - d] + yval[4 + d]);
            }

            // Update result in global memory
            g_u1[idx] = yval[4] + yval[4] - g_u1[idx] + alpha * tmp;
        }

        // Shift yvals up for next iteration
        #pragma unroll 8
        for (int i = 0; i < 8; i++)
        {
            yval[i] = yval[i + 1];
        }

        // Advance global index
        idx += nx;
        __syncthreads();
    }
}