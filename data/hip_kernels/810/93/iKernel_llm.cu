#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void iKernel(float *src, float *dst, int size)
{
    // Calculate the global index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread is within bounds
    if (idx < size) {
        // Perform the computation
        dst[idx] = src[idx] * 2.0f;
    }
}