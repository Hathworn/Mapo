#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling2(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // Unrolling 2; combine pairs of elements if within bounds
    if (idx + blockDim.x < n) {
        g_idata[idx] += g_idata[idx + blockDim.x];
    }

    __syncthreads();

    // Use shared memory to reduce bank conflicts and improve efficiency
    extern __shared__ int shared_data[];
    shared_data[tid] = idata[tid];
    __syncthreads();

    // In-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            shared_data[tid] += shared_data[tid + stride];
        }
        // Synchronize within threadblock
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = shared_data[0];
    }
}