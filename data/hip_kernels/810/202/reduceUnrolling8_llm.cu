#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling8(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n)
    {
        // Load 8 elements per thread and sum them
        int sum = g_idata[idx] + g_idata[idx + blockDim.x] +
                  g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x] +
                  g_idata[idx + 4 * blockDim.x] + g_idata[idx + 5 * blockDim.x] +
                  g_idata[idx + 6 * blockDim.x] + g_idata[idx + 7 * blockDim.x];
        idata[tid] = sum;
    } 
    else {
        idata[tid] = (idx < n) ? g_idata[idx] : 0; // Boundary check
    }

    __syncthreads();

    // in-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads(); // synchronize within threadblock
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}