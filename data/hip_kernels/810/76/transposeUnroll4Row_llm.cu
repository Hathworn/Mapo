#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll4Row(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Precompute common index multiplications
    unsigned int block_offset = blockDim.x * ny;
    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    // Unroll loop with boundary check outside the loop
    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        #pragma unroll
        for (int i = 0; i < 4; i++)
        {
            out[to + i * block_offset] = in[ti + i * blockDim.x];
        }
    }
}