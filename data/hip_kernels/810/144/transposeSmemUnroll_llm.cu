#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BDIMX 32
#define BDIMY 8

__global__ void transposeSmemUnroll(float *out, float *in, const int nx, const int ny)
{
    // Static 1D shared memory
    __shared__ float tile[BDIMY][BDIMX * 2];

    // Coordinate in original matrix
    unsigned int ix = 2 * blockIdx.x * BDIMX + threadIdx.x;
    unsigned int iy = blockIdx.y * BDIMY + threadIdx.y;

    // Linear global memory index for original matrix
    if (ix + BDIMX < nx && iy < ny) 
    {
        // Load two rows from global memory to shared memory
        unsigned int ti = iy * nx + ix;
        tile[threadIdx.y][threadIdx.x]       = in[ti];
        tile[threadIdx.y][threadIdx.x + BDIMX] = in[ti + BDIMX];
    }

    // Thread synchronization
    __syncthreads();
    
    // Coordinate in transposed matrix
    unsigned int ix2 = blockIdx.y * BDIMY + threadIdx.x;
    unsigned int iy2 = 2 * blockIdx.x * BDIMX + threadIdx.y;

    // Linear global memory index for transposed matrix
    if (ix2 < ny && iy2 + BDIMX < nx) 
    {
        // Store two rows to global memory from two columns of shared memory
        unsigned int to = iy2 * ny + ix2;
        out[to] = tile[threadIdx.x][threadIdx.y];
        out[to + ny * BDIMX] = tile[threadIdx.x + BDIMX][threadIdx.y];
    }
}