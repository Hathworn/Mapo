#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_2dfd_last(float *g_u1, float *g_u2, const int nx, const int iStart, const int iEnd) {
    // global to slice : global grid idx for (x,y) plane
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;

    // smem idx for current point
    unsigned int stx = threadIdx.x + NPAD;
    unsigned int idx = ix + iStart * nx;

    // shared memory for u2 with size [4+16+4]
    __shared__ float tile[BDIMX + NPAD2];

    const float alpha = 0.12f;

    // register for y value
    float yval[9];

    for (int i = 0; i < 8; i++)
        yval[i] = g_u2[idx + (i - 4) * nx];

    // to be used in z loop
    int iskip = NPAD * nx;

    #pragma unroll 9
    for (int iy = iStart; iy < iEnd; iy++) {
        // get front3 here
        yval[8] = g_u2[idx + iskip];

        // Load halo regions into shared memory
        if (threadIdx.x < NPAD) {
            tile[threadIdx.x] = g_u2[idx - NPAD];
            tile[stx + BDIMX] = g_u2[idx + BDIMX];
        }

        tile[stx] = yval[4];
        __syncthreads(); // Synchronize threads within block

        if ((ix >= NPAD) && (ix < nx - NPAD)) {
            // 8rd fd operator
            float tmp = coef[0] * tile[stx] * 2.0f;

            #pragma unroll
            for (int d = 1; d <= 4; d++) {
                tmp += coef[d] * (tile[stx - d] + tile[stx + d]);
            }

            #pragma unroll
            for (int d = 1; d <= 4; d++) {
                tmp += coef[d] * (yval[4 - d] + yval[4 + d]);
            }

            // Modified computation: reduce global memory accesses by using registers
            g_u1[idx] = yval[4] + yval[4] - g_u1[idx] + alpha * tmp;
        }

        #pragma unroll 8
        for (int i = 0; i < 8; i++) {
            yval[i] = yval[i + 1];
        }

        // advancd on global idx
        idx += nx;
        __syncthreads();
    }
}