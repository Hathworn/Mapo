#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeNaiveRow(float *out, float *in, const int nx, const int ny)
{
    // Calculate the row and column indices within the grid
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Only execute if within bounds
    if (ix < nx && iy < ny)
    {
        // Perform the transpose, leveraging memory coalescing
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}