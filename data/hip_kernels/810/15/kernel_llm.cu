#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *F, double *D)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Only allow thread 0 in block 0 to execute this
    if (tid == 0 && blockIdx.x == 0)
    {
        *F = 12.1f;  // Use float literal for F
        *D = 12.1;   // Double literal for D
    }
}