#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeOffset(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Optimize by minimizing calculation
    A += i;
    B += i;
    C += k;

    if (k < n) {
        // Use temporary variable to reduce global memory access latency
        float a = *A;
        float b = *B;
        *C = a + b;
    }
}