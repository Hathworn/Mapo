#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check idx within bounds and perform addition only once
    if (idx < N)
    {
        C[idx] = A[idx] + B[idx];
    }
}