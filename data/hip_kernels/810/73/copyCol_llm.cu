#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyCol(float *out, float *in, const int nx, const int ny)
{
    // Use shared memory for better performance
    extern __shared__ float tile[];

    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        // Load input data into shared memory
        tile[threadIdx.y * blockDim.x + threadIdx.x] = in[ix * ny + iy];
        __syncthreads();

        // Copy data from shared memory to output
        out[ix * ny + iy] = tile[threadIdx.y * blockDim.x + threadIdx.x];
    }
}