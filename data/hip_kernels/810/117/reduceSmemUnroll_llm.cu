#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // Static shared memory
    __shared__ int smem[DIM];

    // Set thread ID
    unsigned int tid = threadIdx.x;

    // Global index
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // Unrolling 4 blocks
    int localSum = 0;

    if (idx + 3 * blockDim.x < n)
    {
        // Prefetch global memory to registers
        float a1 = g_idata[idx];
        float a2 = g_idata[idx + blockDim.x];
        float a3 = g_idata[idx + 2 * blockDim.x];
        float a4 = g_idata[idx + 3 * blockDim.x];
        localSum = a1 + a2 + a3 + a4;
    }

    smem[tid] = localSum;
    __syncthreads();

    // In-place reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
    {
        if (tid < s)
        {
            smem[tid] += smem[tid + s];
        }
        __syncthreads();
    }

    // Unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}