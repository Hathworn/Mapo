#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(int *out, int N) {
    // Calculate the global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a single if condition for index within bounds
    if (tid < N) {
        out[tid] = 0;
    }
}