#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceCompleteUnrollWarps8Float (float *g_idata, float *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    float *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // Unrolling 8
    float sum = 0;
    if (idx + 7 * blockDim.x < n)
    {
        sum += g_idata[idx];
        sum += g_idata[idx + blockDim.x];
        sum += g_idata[idx + 2 * blockDim.x];
        sum += g_idata[idx + 3 * blockDim.x];
        sum += g_idata[idx + 4 * blockDim.x];
        sum += g_idata[idx + 5 * blockDim.x];
        sum += g_idata[idx + 6 * blockDim.x];
        sum += g_idata[idx + 7 * blockDim.x];
        idata[tid] = sum;
    }
    else
    {
        if (idx < n) idata[tid] = g_idata[idx];
        else idata[tid] = 0;
    }

    __syncthreads();

    // In-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();

    // Unrolling warp
    if (tid < 32)
    {
        volatile float *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // Write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}