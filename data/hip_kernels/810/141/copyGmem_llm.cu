#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyGmem(float *out, float *in, const int nx, const int ny)
{
    // Using shared memory for faster access
    extern __shared__ float tile[];

    // Matrix coordinate (ix, iy)
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;

    // Load data into shared memory with boundary test
    if (ix < nx && iy < ny)
    {
        tile[threadIdx.y * blockDim.x + threadIdx.x] = in[idx];
    }
    __syncthreads();

    // Write back from shared memory to global memory
    if (ix < nx && iy < ny)
    {
        out[idx] = tile[threadIdx.y * blockDim.x + threadIdx.x];
    }
}