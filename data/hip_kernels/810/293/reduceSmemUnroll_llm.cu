#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // Allocate shared memory
    __shared__ int smem[DIM];

    // Calculate thread ID
    unsigned int tid = threadIdx.x;

    // Calculate global index, process 4 blocks of input data at a time
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // Initialize local sum
    int tmpSum = 0;

    // Only add within boundary
    if (idx < n) tmpSum += g_idata[idx];
    if (idx + blockDim.x < n) tmpSum += g_idata[idx + blockDim.x];
    if (idx + 2 * blockDim.x < n) tmpSum += g_idata[idx + 2 * blockDim.x];
    if (idx + 3 * blockDim.x < n) tmpSum += g_idata[idx + 3 * blockDim.x];

    // Store result in shared memory
    smem[tid] = tmpSum;
    __syncthreads();

    // Perform in-place reduction within shared memory
    if (blockDim.x >= 1024) {
        if(tid < 512) smem[tid] += smem[tid + 512];
        __syncthreads();
    }

    if (blockDim.x >= 512) {
        if(tid < 256) smem[tid] += smem[tid + 256];
        __syncthreads();
    }

    if (blockDim.x >= 256) {
        if(tid < 128) smem[tid] += smem[tid + 128];
        __syncthreads();
    }

    if (blockDim.x >= 128) {
        if(tid < 64) smem[tid] += smem[tid + 64];
        __syncthreads();
    }

    // Reduce within warp using volatile
    if (tid < 32) {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // Write block result to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}