#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmem(int *g_idata, int *g_odata, unsigned int n)
{
    __shared__ int smem[DIM];

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    int *idata = g_idata + blockIdx.x * blockDim.x;
    smem[tid] = idata[tid];
    __syncthreads();

    // Loop unrolling and combined reductions to reduce syncthreads
    if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];
    __syncthreads();
    
    if (blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];
    __syncthreads();

    // Unroll last warp using thread divergence avoidance
    if (tid < 32) {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        __syncthreads();
        
        vsmem[tid] += vsmem[tid + 16];
        __syncthreads();
        
        vsmem[tid] += vsmem[tid + 8];
        __syncthreads();
        
        vsmem[tid] += vsmem[tid + 4];
        __syncthreads();
        
        vsmem[tid] += vsmem[tid + 2];
        __syncthreads();
        
        vsmem[tid] += vsmem[tid + 1];
        __syncthreads();
    }

    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}