#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceInterleaved(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if (idx >= n) return;

    // In-place reduction in global memory using shared memory for improved performance
    extern __shared__ int sdata[];
    sdata[tid] = idata[tid];
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            sdata[tid] += sdata[tid + stride];
        }
        
        __syncthreads(); // Ensure all additions are completed before next iteration
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}