#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceCompleteUnrollWarps8 (int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // Unroll and sum 8 elements at once if within bounds
    if (idx + 7 * blockDim.x < n)
    {
        int sum = 0;
        for (int i = 0; i < 8; i++)
        {
            sum += g_idata[idx + i * blockDim.x]; // Efficient unrolling with loop
        }
        g_idata[idx] = sum;
    }

    __syncthreads();

    // In-place reduction using loop for simplicity and readability
    for (int s = blockDim.x / 2; s > 32; s >>= 1)
    {
        if (tid < s) 
        {
            idata[tid] += idata[tid + s];
        }
        __syncthreads();
    }

    // Unrolling warp for efficiency
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}