#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeOffsetUnroll2(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Ensure the loop is fully unrolled for better performance
    if (k < n)
    {
        C[k] = A[i] + B[i];
        
        // Check and process remaining elements
        if (k + blockDim.x < n)
        {
            C[k + blockDim.x] = A[i + blockDim.x] + B[i + blockDim.x];
        }
    }
}