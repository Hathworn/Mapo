#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmemDyn(int *g_idata, int *g_odata, unsigned int n)
{
    extern __shared__ int smem[];

    // Set thread ID
    unsigned int tid = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Set to smem by each thread
    smem[tid] = idata[tid];
    __syncthreads();

    // In-place reduction in shared memory with loop unrolling
    if (blockDim.x >= 1024) { if (tid < 512) smem[tid] += smem[tid + 512]; __syncthreads(); }
    if (blockDim.x >= 512)  { if (tid < 256) smem[tid] += smem[tid + 256]; __syncthreads(); }
    if (blockDim.x >= 256)  { if (tid < 128) smem[tid] += smem[tid + 128]; __syncthreads(); }
    if (blockDim.x >= 128)  { if (tid < 64)  smem[tid] += smem[tid + 64];  __syncthreads(); }
    
    // Unrolling warp - optimized version without volatile
    if (tid < 32)
    {
        smem[tid] += smem[tid + 32];
        smem[tid] += smem[tid + 16];
        smem[tid] += smem[tid + 8];
        smem[tid] += smem[tid + 4];
        smem[tid] += smem[tid + 2];
        smem[tid] += smem[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}