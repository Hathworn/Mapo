#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll4Col(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = (blockDim.x * blockIdx.x + threadIdx.x) * 4;  // Unroll adjustment
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        unsigned int ti = iy * nx + ix; // access in rows
        unsigned int to = ix * ny + iy; // access in columns
        
        // Simplify by using variables for offsets
        float in0 = in[to];
        float in1 = in[to + ny];
        float in2 = in[to + 2 * ny];
        float in3 = in[to + 3 * ny];
        
        out[ti]                = in0;
        out[ti +   blockDim.x] = in1;
        out[ti + 2 * blockDim.x] = in2;
        out[ti + 3 * blockDim.x] = in3;
    }
}