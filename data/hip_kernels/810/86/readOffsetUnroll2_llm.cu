#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffsetUnroll2(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    unsigned int k = i + offset;

    // Use a local shared memory block for coalesced memory access
    extern __shared__ float sharedMem[];
    float *sharedA = sharedMem;
    float *sharedB = sharedA + blockDim.x * 2;

    if (k + blockDim.x < n)
    {
        // Load data into shared memory
        sharedA[threadIdx.x] = A[k];
        sharedB[threadIdx.x] = B[k];
        sharedA[threadIdx.x + blockDim.x] = A[k + blockDim.x];
        sharedB[threadIdx.x + blockDim.x] = B[k + blockDim.x];
        __syncthreads();

        // Compute using shared memory
        C[i] = sharedA[threadIdx.x] + sharedB[threadIdx.x];
        C[i + blockDim.x] = sharedA[threadIdx.x + blockDim.x] + sharedB[threadIdx.x + blockDim.x];
    }
}