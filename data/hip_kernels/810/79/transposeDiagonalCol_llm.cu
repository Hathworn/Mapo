#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeDiagonalCol(float *out, float *in, const int nx, const int ny)
{
    unsigned int blk_y = blockIdx.y; // Use blockIdx.y directly for clarity
    unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    // Combine boundary check in one line for better performance
    if (ix < nx && iy < ny)
    {
        // Swap ix and iy in accessing 'in' to handle column-major access more efficiently
        out[iy * nx + ix] = in[iy * ny + ix];
    }
}