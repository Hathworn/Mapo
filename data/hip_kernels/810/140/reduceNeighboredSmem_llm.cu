#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighboredSmem(int *g_idata, int *g_odata, unsigned int n)
{
    extern __shared__ int smem[]; // Use dynamic shared memory

    // Set thread ID and block starting index
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (idx >= n) return;

    // Load elements into shared memory with bounds check
    smem[tid] = (tid < blockDim.x) ? g_idata[idx] : 0;
    __syncthreads();

    // In-place reduction using shared memory
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        int index = 2 * stride * tid;

        if (index < blockDim.x)
        {
            smem[index] += smem[index + stride];
        }
        
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}