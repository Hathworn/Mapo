#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysZeroCopyOffset(float *A, float *B, float *C, const int N, int offset)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + offset;

    // Unroll loop to improve performance
    if (i - offset < N) {
        C[i] = A[i] + B[i];
    }
}