#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyRow(float *out, float *in, const int nx, const int ny)
{
    // Calculate thread's global row and column indices
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Use shared memory to optimize memory access
    __shared__ float tile[32][32]; // Assuming blockDim.x and blockDim.y are both 32

    if (ix < nx && iy < ny)
    {
        // Read from global memory into shared memory
        tile[threadIdx.y][threadIdx.x] = in[iy * nx + ix];
        __syncthreads();

        // Write back to global memory from shared memory
        out[iy * nx + ix] = tile[threadIdx.y][threadIdx.x];
    }
}