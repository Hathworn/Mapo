#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighbored (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    // in-place reduction in global memory
    // Use warp shuffle to avoid shared memory and synchronization
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        int temp = tid + stride < blockDim.x ? idata[tid + stride] : 0;
        idata[tid] += temp;

        // improve performance: avoid unnecessary synchronization in warp
        if (stride < 32) 
        {
            __syncwarp();
        } 
        else 
        {
            __syncthreads();
        }
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}