#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeOffset(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Use shared memory if possible for increased performance on larger datasets
    extern __shared__ float sharedMemory[];
    float *sharedA = &sharedMemory[0];
    float *sharedB = &sharedMemory[blockDim.x];

    if (i < n) {
        // Load data into shared memory
        sharedA[threadIdx.x] = A[i];
        sharedB[threadIdx.x] = B[i];
    }
    __syncthreads();

    if (k < n) {
        // Perform the operation from shared memory to optimize memory access
        C[k] = sharedA[threadIdx.x] + sharedB[threadIdx.x];
    }
}