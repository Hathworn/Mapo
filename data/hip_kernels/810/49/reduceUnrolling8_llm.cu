#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling8 (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8 and optimization by using shared memory
    __shared__ int sharedData[8 * blockDim.x];

    if (idx + 7 * blockDim.x < n)
    {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        int b1 = g_idata[idx + 4 * blockDim.x];
        int b2 = g_idata[idx + 5 * blockDim.x];
        int b3 = g_idata[idx + 6 * blockDim.x];
        int b4 = g_idata[idx + 7 * blockDim.x];
        sharedData[tid] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }
    else
    {
        sharedData[tid] = 0;
    }

    __syncthreads();

    // reduce in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            sharedData[tid] += sharedData[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sharedData[0];
}