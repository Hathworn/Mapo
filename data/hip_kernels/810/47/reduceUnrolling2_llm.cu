#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling2(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // Unrolling 2
    int temp = 0;
    if (idx < n) temp = idata[tid];
    if (idx + blockDim.x < n) temp += idata[tid + blockDim.x]; 

    __syncthreads();

    // In-place reduction using shared memory
    __shared__ int sdata[1024]; // Assuming blockDim.x <= 1024
    sdata[tid] = temp;
    __syncthreads();

    // Parallel reduction with shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}