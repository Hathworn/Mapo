#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; // Calculate x index
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y; // Calculate y index
    unsigned int idx = iy * nx + ix;                         // Calculate global index

    if (ix < nx && iy < ny)  // Bound checking for matrix dimensions
        MatC[idx] = MatA[idx] + MatB[idx];  // Matrix addition
}