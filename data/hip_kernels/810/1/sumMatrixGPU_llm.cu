#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny) {
        // Load matrix elements to shared memory to exploit memory coalescing
        __shared__ float sharedMatA[32][32];
        __shared__ float sharedMatB[32][32];

        sharedMatA[threadIdx.y][threadIdx.x] = MatA[idx];
        sharedMatB[threadIdx.y][threadIdx.x] = MatB[idx];

        __syncthreads(); // Ensure all elements are loaded before computation

        // Write the result to global memory
        MatC[idx] = sharedMatA[threadIdx.y][threadIdx.x] + sharedMatB[threadIdx.y][threadIdx.x];
    }
}