#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate global thread indices
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    // Check for boundary conditions and perform addition
    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}