#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readWriteOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate index using offset
    unsigned int k = blockIdx.x * blockDim.x + threadIdx.x + offset;

    // Ensure computation only occurs if within bounds
    if (k < n) {
        // Perform the computation directly if within bounds
        C[k] = A[k] + B[k];
    }
}