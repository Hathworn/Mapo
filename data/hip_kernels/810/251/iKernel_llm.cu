#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use restrict keyword for pointers to improve memory access performance
__global__ void iKernel(float * __restrict__ A, float * __restrict__ B, float * __restrict__ C, const int N)
{
    // Cache the index calculation to avoid recalculation
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Reorganize logic to remove branch divergence
    if (i < N) 
    { 
        C[i] = A[i] + B[i]; 
    }
}