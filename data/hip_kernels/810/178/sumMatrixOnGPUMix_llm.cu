#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = blockIdx.y;
    unsigned int idx = iy * nx + ix;

    // Check if x and y coordinates are within matrix dimensions
    if (iy < ny && ix < nx)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}