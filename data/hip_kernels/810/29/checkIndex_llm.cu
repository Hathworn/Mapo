#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void checkIndex(void)
{
    // Combine all output into one printf to reduce I/O overhead
    printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) gridDim:(%d, %d, %d)\n", 
           threadIdx.x, threadIdx.y, threadIdx.z, 
           blockIdx.x, blockIdx.y, blockIdx.z,
           blockDim.x, blockDim.y, blockDim.z,
           gridDim.x, gridDim.y, gridDim.z);
}