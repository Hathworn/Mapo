#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    // Cache the index calculation to improve performance
    unsigned int k = blockIdx.x * blockDim.x + threadIdx.x + offset;

    // Check if within bounds before performing operations
    if (k < n) {
        C[k - offset] = A[k] + B[k]; // Use calculated index directly
    }
}