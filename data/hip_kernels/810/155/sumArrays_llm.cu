#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the computation is performed only for idx < N
    if (idx < N)
    {
        C[idx] = A[idx] + B[idx];
    }
}