#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyCol(float *out, float *in, const int nx, const int ny)
{
    // Calculate linear index for improved memory access
    unsigned int idx = blockIdx.y * blockDim.y * nx + blockIdx.x * blockDim.x + threadIdx.y * nx + threadIdx.x;

    if (blockIdx.x * blockDim.x + threadIdx.x < nx && blockIdx.y * blockDim.y + threadIdx.y < ny)
    {
        // Efficiently copy the element using linear index
        out[idx] = in[idx];
    }
}