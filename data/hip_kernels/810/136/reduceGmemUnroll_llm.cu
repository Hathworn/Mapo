#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceGmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x * 4;

    // Unroll 4 - efficient memory access
    int sum = 0;
    if (idx < n) sum += g_idata[idx];
    if (idx + blockDim.x < n) sum += g_idata[idx + blockDim.x];
    if (idx + 2 * blockDim.x < n) sum += g_idata[idx + 2 * blockDim.x];
    if (idx + 3 * blockDim.x < n) sum += g_idata[idx + 3 * blockDim.x];
    idata[tid] = sum;

    __syncthreads();

    // In-place reduction using loop - remove redundancy
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            idata[tid] += idata[tid + s];
        }
        __syncthreads();
    }

    // Warp unrolling - reduce sync overhead
    if (tid < 32) {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}