#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TIME                5.
#define TIME_STEP           .1

#define STEP                1.
#define K                   TIME_STEP / SQUARE(STEP)

#define SQUARE(x)           (x * x)
#define HANDLE_ERROR(err)   (HandleError(err, __FILE__, __LINE__))

__global__ void Kernel(double * device, const uint size)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use shared memory for temporary storage if applicable
    __shared__ double temp[blockDim.x];
    
    // Ensure processing within bounds
    if (i >= size) return;

    double left, center, right;
    if (i > 0) {
        left = device[i - 1];
    }
    
    center = device[i];

    if (i < size - 1) {
        right = device[i + 1];
    }

    // Reduce conditional check; compute values based on valid indices
    if (i == 0) {
        device[i] = .0;
    } else if (i == size - 1) {
        device[size - 1] = device[size - 2] + 5 * STEP;
    } else {
        // Apply finite difference method
        device[i] = (right - 2.0 * center + left) * K + center;
    }
}