#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    // Calculate the unique thread index within the total number of threads
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Calculate output pixel coordinates and channel indices
    int out_index = i;
    int out_w = (out_index / c) % (w * stride);
    int out_h = (out_index / (w * stride * c)) % (h * stride);
    int out_c = out_index % c;
    int b = out_index / (w * stride * h * stride * c);

    // Calculate the corresponding input pixel coordinates
    int in_w = out_w / stride;
    int in_h = out_h / stride;

    // Calculate the input index
    int in_index = b * w * h * c + out_c * w * h + in_h * w + in_w;

    // Perform the forward or backward pass
    if (forward) {
        out[out_index] += scale * x[in_index];
    } else {
        atomicAdd(x + in_index, scale * out[out_index]);
    }
}