#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // More efficient index calculation
    int stride = gridDim.x * blockDim.x;  // Determine stride for grid-stride loop
    for (int i = idx; i < n; i += stride) {  // Grid-stride loop for better utilization
        float t = truth[i];
        float p = pred[i];
        error[i] = -t*logf(p + 1e-7f) - (1 - t)*logf(1 - p + 1e-7f);  // Replacing log with logf & adjusted constant
        delta[i] = t - p;
    }
}