#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use an improved indexing method to enhance performance.
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop over grid to handle larger arrays.
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        X[idx * INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[idx * INCX]));
    }
}