#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        // Store the difference value to avoid redundant memory accesses
        float diff = truth[i] - pred[i];
        // Error calculation optimized using single assignment
        error[i] = diff * diff;
        delta[i] = diff;
    }
}