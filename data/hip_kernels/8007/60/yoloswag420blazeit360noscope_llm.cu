#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define UNROLL_FACTOR 4  // define unroll factor

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int id_end = size - id; // calculate remaining threads
    int unroll_limit = id_end < blockDim.x*UNROLL_FACTOR ? id_end : blockDim.x*UNROLL_FACTOR; // limit unrolling to remaining size

    // Unrolling loop
    for (int i = 0; i < unroll_limit; i += blockDim.x)
    {
        int idx = id + i;
        if (idx < size) 
        {
            input[idx] = (rand[idx] < prob) ? 0 : input[idx]*scale;
        }
    }
}