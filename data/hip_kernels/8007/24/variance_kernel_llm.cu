#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1.f/(batch * spatial - 1);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i >= filters) 
        return;

    // Using shared memory for variance to improve memory access efficiency
    __shared__ float shared_variance[1024];
    shared_variance[threadIdx.x] = 0;

    for(int j = 0; j < batch; ++j){
        for(int k = 0; k < spatial; ++k){
            int index = j * filters * spatial + i * spatial + k;
            shared_variance[threadIdx.x] += powf((x[index] - mean[i]), 2);
        }
    }

    __syncthreads();

    // Atomic addition to prevent race conditions
    atomicAdd(&variance[i], shared_variance[threadIdx.x] * scale);
}