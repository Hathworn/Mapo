#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
  // Calculate unique thread index
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // Process the array with a stride of total number of threads
  for (; i < n; i += blockDim.x * gridDim.x) {
    binary[i] = (x[i] >= 0) ? 1 : -1;
  }
}