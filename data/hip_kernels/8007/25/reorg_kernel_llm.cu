#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    int in_w = i % w;
    int in_h = (i / w) % h;
    int in_c = (i / (w * h)) % c;
    int b = i / (w * h * c);
    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    if (forward)
        out[out_index] = x[i];
    else
        out[i] = x[out_index];
}