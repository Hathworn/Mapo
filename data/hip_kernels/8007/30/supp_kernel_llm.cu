#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use a single-dimensional kernel for better memory coalescing and simpler indexing
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i < N) {
        // Cache X[i*INCX] value to avoid redundant global memory access
        float val = X[i*INCX];
        if((val * val) < (ALPHA * ALPHA)) {
            X[i*INCX] = 0;
        }
    }
}