#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Calculate global thread index.
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    
    // Exit if f2 <= f1 as per original logic.
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;

    // Optimize loop structure for memory coalescing and avoid recalculating indices.
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int offset = b * size * n + i;
            int i1 = offset + f1 * size;
            int i2 = offset + f2 * size;
            float val1 = output[i1];
            float val2 = output[i2];
            sum += val1 * val2;
            norm1 += val1 * val1;
            norm2 += val2 * val2;
        }
    }

    // Compute final normalized sum.
    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    sum = sum / norm;

    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int offset = b * size * n + i;
            int i1 = offset + f1 * size;
            int i2 = offset + f2 * size;
            float val1 = output[i1];
            float val2 = output[i2];
            float scaled_sum = -scale * sum / norm;
            delta[i1] += scaled_sum * val2;
            delta[i2] += scaled_sum * val1;
        }
    }
}