#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * gridDim.x * blockDim.x); 

    // Ensure thread index is within bounds
    if(i < (NX + NY) * B) {
        int b = i / (NX + NY);
        int j = i % (NX + NY);

        if (j < NX){
            if(X) X[b * NX + j] += OUT[i];
        } else {
            if(Y) Y[b * NY + (j - NX)] += OUT[i];
        }
    }
}