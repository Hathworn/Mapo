#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Use shared memory to reduce global memory access latency
    extern __shared__ float shared_weights[];
    float mean = 0;

    for(int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[i] = fabsf(weights[f*size + i]);
    }
    __syncthreads();

    for(int i = 0; i < size; ++i) {
        mean += shared_weights[i];
    }
    mean = mean / size;

    for(int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
    }
}