#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int c = in_c;

    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x; 
    if (id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    int out_index = j + w * (i + h * (k + c * b)); 
    float max = -INFINITY;
    int max_i = -1;

    // Loop unrolling for improved performance
    for (int l = 0; l < size; ++l) {
        int cur_h = h_offset + i * stride + l;
        for (int m = 0; m < size; ++m) {
            int cur_w = w_offset + j * stride + m;
            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            int valid = (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w);
            float val = valid ? input[index] : -INFINITY;
            if (val > max) {
                max = val;
                max_i = index;
            }
        }
    }

    output[out_index] = max;
    indexes[out_index] = max_i;
}