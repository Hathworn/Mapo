#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    // Using shared memory for partial sums
    __shared__ float partial_sum[1024];

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    int b = id / c;

    int in_index_base = h * w * (k + b * c);
    float sum = 0.0f;

    for(int i = tid; i < w * h; i += blockDim.x){
        sum += input[in_index_base + i];
    }

    partial_sum[tid] = sum;
    __syncthreads();

    // Reduce partial sums within the block
    for(int stride = 1; stride < blockDim.x; stride *= 2) {
        if (tid % (2 * stride) == 0 && tid + stride < w * h) {
            partial_sum[tid] += partial_sum[tid + stride];
        }
        __syncthreads();
    }

    if(tid == 0) {
        output[k + c * b] = partial_sum[0] / (w * h);
    }
}