#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify calculation of index

    if(i < N) { // Ensure thread doesn't exceed array boundaries
        X[i * INCX] = ALPHA; // Assign ALPHA to each element of the array
    }
}