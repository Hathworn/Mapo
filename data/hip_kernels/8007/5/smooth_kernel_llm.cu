#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -(size / 2.f);
    int h_offset = -(size / 2.f);

    int out_index = j + w * (i + h * (k + c * b));
    int valid, cur_h, cur_w, index;
    float diff;

    for (int l = 0; l < size; ++l)
    {
        for (int m = 0; m < size; ++m)
        {
            cur_h = h_offset + i + l;
            cur_w = w_offset + j + m;
            valid = (cur_h >= 0 && cur_h < h && cur_w >= 0 && cur_w < w);
            if (valid)
            {
                index = cur_w + w * (cur_h + h * (k + c * b));
                diff = x[index] - x[out_index];
                delta[out_index] += rate * diff;
            }
        }
    }
}