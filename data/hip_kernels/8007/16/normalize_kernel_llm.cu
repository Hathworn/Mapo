#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    // Pre-fetch mean and variance to avoid redundant global memory accesses
    int f = (index / spatial) % filters;
    float m = mean[f];
    float v = variance[f];
    
    x[index] = (x[index] - m) / sqrtf(v + 0.00001f); // Normalize
}