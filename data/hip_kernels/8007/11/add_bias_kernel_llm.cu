#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    // Using flat thread index for direct computation
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // iterate through the elements step by step with stride equal to the total number of threads
    int step = blockDim.x * gridDim.x;

    // Processing multiple elements by incremental step stride
    for (int index = idx; index < n * size * batch; index += step)
    {
        int i = index % size;
        int j = (index / size) % n;
        int k = index / (size * n);

        output[(k * n + j) * size + i] += biases[j];
    }
}