#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (i < (NX + NY) * B) {
        int b = i / (NX + NY);
        int j = i % (NX + NY);

        // Choose the correct source array based on the value of j
        OUT[i] = (j < NX) ? X[b * NX + j] : Y[b * NY + j - NX];
    }
}