#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation
    if (idx < N) X[idx * INCX] += ALPHA; // Shortened variable name for better readability
}