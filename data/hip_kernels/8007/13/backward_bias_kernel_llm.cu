#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int i, b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Reduce work by accumulating across batches
    for (b = 0; b < batch; ++b) {
        for (i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index];
        }
    }

    part[p] = sum;
    __syncthreads();

    // Reduce sum within the block
    for (i = BLOCK / 2; i > 0; i >>= 1) {
        if (p < i) {
            part[p] += part[p + i];
        }
        __syncthreads();
    }

    // Update bias only by thread 0 of each block
    if (p == 0) {
        atomicAdd(&bias_updates[filter], part[0]);
    }
}