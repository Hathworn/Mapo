#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation
    int gridStride = gridDim.x * blockDim.x; // Calculate grid stride
    while (i < N) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
        i += gridStride; // Iterate through equally spaced grid indices
    }
}