#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int f = (index/spatial)%filters;

    // Precompute often used terms
    float inv_sqrt_variance = 1.f / sqrtf(variance[f] + .00001f);
    float two_variance_delta = 2.f * variance_delta[f] / (spatial * batch);
    float mean_delta_term = mean_delta[f] / (spatial * batch);
    float mean_diff = x[index] - mean[f];

    // Perform the delta normalization
    delta[index] = delta[index] * inv_sqrt_variance + two_variance_delta * mean_diff + mean_delta_term;
}