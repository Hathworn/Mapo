#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int c = in_c;
    int area = (size - 1) / stride;

    int id = blockIdx.x * blockDim.x + threadIdx.x; // Compute 1D index
    if (id >= n) return;

    int j = id % in_w;
    int i = (id / in_w) % in_h;
    int k = (id / (in_w * in_h)) % in_c;
    int b = id / (in_c * in_h * in_w);

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    float d = 0.0f;
    int out_w = 0, out_h = 0, out_index = 0, valid = 0;

    for (int l = -area; l <= area; ++l) {
        for (int m = -area; m <= area; ++m) {
            out_w = (j - w_offset) / stride + m;
            out_h = (i - h_offset) / stride + l;
            out_index = out_w + w * (out_h + h * (k + c * b));
            valid = (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h);
            d += (valid && indexes[out_index] == id) ? delta[out_index] : 0.0f; // Simplified condition and accumulation
        }
    }
    prev_delta[id] += d; // Update prev_delta once
}