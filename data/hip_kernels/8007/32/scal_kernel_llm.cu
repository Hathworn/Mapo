#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for better performance
    for (; i < N; i += blockDim.x * gridDim.x) {
        X[i * INCX] *= ALPHA;
    }
}