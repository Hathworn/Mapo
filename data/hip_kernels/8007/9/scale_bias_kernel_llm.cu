#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    // Calculate global thread index
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Check bounds and apply bias scaling
    if(offset < size) {
        int index = (batch * n + filter) * size + offset; // Precompute index to prevent repetitive calculation
        output[index] *= biases[filter];
    }
}