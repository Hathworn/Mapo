#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation
    if (i < n && mask[i] == mask_num)  // Conditional to check within bounds and mask number
    {
        x[i] = val;  // Assign value if condition meets
    }
}