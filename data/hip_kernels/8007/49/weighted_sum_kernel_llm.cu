#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate the thread's global index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Ensure the index is within bounds
    if(i < n) {
        float b_value = (b != nullptr) ? b[i] : 0;
        c[i] = s[i] * a[i] + (1.0f - s[i]) * b_value;
    }
}