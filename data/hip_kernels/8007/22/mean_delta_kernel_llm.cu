#include "hip/hip_runtime.h"
```
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique thread index
    if (i >= filters) return;

    float md = 0.0f; // Initialize mean_delta

    // Use a single loop to iterate over all indices
    for (int j = 0; j < batch * spatial; ++j) {
        int index = (j / spatial) * filters * spatial + i * spatial + (j % spatial);
        md += delta[index];
    }
    mean_delta[i] = md * (-1.0f / sqrtf(variance[i] + .00001f)); // Update mean_delta with scaling
}