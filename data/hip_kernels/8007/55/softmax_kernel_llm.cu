#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output) {
    float sum = 0.0f;
    float largest = -INFINITY;

    // Find the largest value in the input
    for (int i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = (val > largest) ? val : largest;
    }

    // Compute the exponentials and sum them up
    for (int i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp);
        sum += e;
        output[i * stride] = e; // Store not normalized value
    }

    // Normalize the exponentials
    float inv_sum = 1.0f / sum; // Precompute the reciprocal of sum
    for (int i = 0; i < n; ++i) {
        output[i * stride] *= inv_sum; // Normalize each value
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= batch * groups) return;
    int b = id / groups;
    int g = id % groups;

    // Compute the offset and apply softmax
    softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}