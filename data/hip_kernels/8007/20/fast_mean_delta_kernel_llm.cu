#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Unroll the loop to increase performance
    for(int j = 0; j < batch; ++j) {
        for(int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += delta[index];
        }
    }
    
    __syncthreads();

    // Reduce within shared memory
    for(int stride = threads / 2; stride > 0; stride >>= 1) {
        if(id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    if(id == 0){
        mean_delta[filter] = local[0] * (-1.f / sqrtf(variance[filter] + .00001f));
    }
}