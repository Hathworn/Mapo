#include "hip/hip_runtime.h"
```
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= n) return;

    extern __shared__ float shared_sum[];
    
    shared_sum[threadIdx.x] = 0;

    for(int b = 0; b < batch; ++b){
        int i = b * n + index;
        shared_sum[threadIdx.x] += delta[i];
    }

    __syncthreads();

    if(threadIdx.x == 0){
        float sum = 0;
        for(int i = 0; i < blockDim.x; ++i){
            sum += shared_sum[i];
        }
        atomicAdd(&bias_updates[index], sum);
    }
}