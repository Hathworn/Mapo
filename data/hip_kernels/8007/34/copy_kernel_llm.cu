#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_kernel(int N,  float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop over grid stride
    for(int index = i; index < N; index += gridDim.x * blockDim.x)
    {
        Y[index * INCY + OFFY] = X[index * INCX + OFFX];
    }
}