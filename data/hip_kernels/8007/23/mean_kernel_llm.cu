#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Optimize scale calculation by moving it outside the kernel
    float scale = 1.f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Boundary check
    if (i >= filters) return;
    
    // Use shared memory to reduce global memory access
    extern __shared__ float shared[];
    
    int j, k;
    float sum = 0;
    for (j = 0; j < batch; ++j) {
        for (k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += x[index]; // Accumulate sum in local variable
        }
    }
    
    mean[i] = sum * scale; // Store the result after multiplication with scale
}