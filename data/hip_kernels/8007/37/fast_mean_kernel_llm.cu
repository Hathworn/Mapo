#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Loop unrolling for improved performance
    for(int j = 0; j < batch; ++j){
        for(int i = 0; i < spatial; i += threads * 2){
            int index1 = j*spatial*filters + filter*spatial + i + id;
            int index2 = j*spatial*filters + filter*spatial + i + threads + id;
            local[id] += (i + id < spatial) ? x[index1] : 0;
            local[id] += (i + threads + id < spatial) ? x[index2] : 0;
        }
    }

    __syncthreads();

    if(id < 32) {  // Warp-level reduction for better efficiency
        for(int i = id + 32; i < threads; i += 32){
            local[id] += local[i];
        }
    }

    __syncthreads();

    if(id == 0) {
        float sum = 0;
        for(int i = 0; i < 32; ++i){
            sum += local[i];
        }
        mean[filter] = sum / (spatial * batch);
    }
}