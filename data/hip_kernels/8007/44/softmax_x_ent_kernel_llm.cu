#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified grid-stride calculation

    // Use stride loop to handle more elements in each thread
    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x) {
        float t = truth[idx];
        float p = pred[idx];
        error[idx] = (t) ? -log(p) : 0;
        delta[idx] = t - p;
    }
}