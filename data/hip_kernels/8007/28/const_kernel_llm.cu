#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Using a stride loop to cover all elements
    for (; i < N; i += stride) {
        X[i * INCX] = ALPHA;
    }
}