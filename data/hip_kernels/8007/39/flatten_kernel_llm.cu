#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified thread index calculation
    if (i >= N) return;

    int b = i / (spatial * layers);  // Compute batch index
    int rem = i % (spatial * layers);  // Remaining index after batch calculation
    int in_c = rem / spatial;  // Compute layer index
    int in_s = rem % spatial;  // Compute spatial index
    
    // Compute direct index for input and output
    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;

    // Forward or backward based on 'forward' flag
    if (forward) 
        out[i2] = x[i1];
    else 
        out[i1] = x[i2];
}