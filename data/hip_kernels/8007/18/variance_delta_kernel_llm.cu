#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Kernel index based on global thread ID
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    // Return if index is out of filter range
    if (i >= filters) return;

    float v_delta = 0.0f;  // Local variable for variance delta
    float local_mean = mean[i];  // Cache mean value
    float local_variance = variance[i];  // Cache variance value

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            v_delta += delta[index] * (x[index] - local_mean);  // Accumulate variance delta
        }
    }

    v_delta *= -.5f * powf(local_variance + .00001f, -1.5f);  
    variance_delta[i] = v_delta;
}