#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Each thread handles multiple elements to maximize parallelism
    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x) {
        float diff = truth[idx] - pred[idx];
        error[idx] = abs(diff);
        delta[idx] = (diff > 0) ? 1 : -1;
    }
}