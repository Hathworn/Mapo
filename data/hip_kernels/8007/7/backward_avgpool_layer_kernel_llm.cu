#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified 1D indexing
    if (id >= n) return;

    int b = id / c;
    int k = id % c;

    float out_delta_val = out_delta[k + c * b];  // Preload out_delta value for reuse
    int stride = h * w;
    int base_in_index = b * c * stride + k * stride;

    for (int i = 0; i < w * h; ++i) {
        atomicAdd(&in_delta[base_in_index + i], out_delta_val / (w * h)); // Use atomicAdd for concurrent writing
    }
}