#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified id calculation
    if (id >= size) return;

    int temp = id;
    int i = temp % minw;
    temp /= minw;
    int j = temp % minh;
    temp /= minh;
    int k = temp % minc;
    temp /= minc;
    int b = temp % batch;

    int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
    int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));
    
    float out_val = out[out_index]; // Load once
    out[out_index] = s1 * out_val + s2 * add[add_index]; // Compute with loaded values
}