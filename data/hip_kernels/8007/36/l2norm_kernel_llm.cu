#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    
    int b = index / spatial;
    int i = index % spatial;
    int f;
    float sum = 0;

    // Pre-calculated base index
    int base_index = b * filters * spatial + i;

    for(f = 0; f < filters; ++f) {
        sum += powf(x[base_index + f * spatial], 2);
    }

    sum = sqrtf(sum);
    if(sum == 0) sum = 1;

    for(f = 0; f < filters; ++f) {
        int idx = base_index + f * spatial;  // Use pre-calculated index
        x[idx] /= sum;
        dx[idx] = (1 - x[idx]) / sum;
    }
}