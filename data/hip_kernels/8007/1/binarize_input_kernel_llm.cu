#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;
    
    // Use shared memory to reduce global memory accesses
    extern __shared__ float shared_mean[];
    float mean = 0;

    for (int i = 0; i < n; ++i) {
        mean += fabsf(input[i * size + s]);
    }
    mean = mean / n;
    shared_mean[threadIdx.x] = mean;

    __syncthreads();

    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? shared_mean[threadIdx.x] : -shared_mean[threadIdx.x];
    }
}