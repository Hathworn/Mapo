#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Each thread processes multiple elements
    int idx = (blockIdx.x * blockDim.x + threadIdx.x);
    int stride = gridDim.x * blockDim.x; // Calculate stride to process larger arrays evenly

    for (int i = idx; i < N; i += stride)
    {
        Y[i * INCY] = pow(X[i * INCX], ALPHA); // Process elements
    }
}