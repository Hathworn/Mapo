#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        float dci = dc[i]; // Load dc[i] into a register for reuse
        float si = s[i];   // Load s[i] into a register for reuse
        if (da) da[i] += dci * si;
        if (db) db[i] += dci * (1 - si);
        ds[i] += dci * (a[i] - b[i]);
    }
}