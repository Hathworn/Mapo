#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;

    // Find the largest value for numerical stability
    for (i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = fmaxf(val, largest);
    }

    // Compute the exponentials and their sum
    for (i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp);
        sum += e;
        output[i * stride] = e;
    }

    // Normalize the output
    for (i = 0; i < n; ++i) {
        output[i * stride] /= sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;

    int s = id % spatial;
    id /= spatial;
    int g = id % groups;
    int b = id / groups;
    int goff = group_offset[g] * spatial;
    int boff = b * stride;

    // Launch softmax computation for each group
    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}