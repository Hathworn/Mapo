#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;

    // Precompute constants for optimization
    float B1_t = 1.f - powf(B1, t);
    float B2_t = 1.f - powf(B2, t);

    float mhat = m[index] / B1_t;
    float vhat = v[index] / B2_t;

    // Use compiler intrinsic for better performance
    x[index] = x[index] + rate * mhat / (__fsqrt_rn(vhat) + eps);
}