#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void wgan_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (i < n) {
        // Precompute reuseable value
        float t = truth[i];
        error[i] = t ? -pred[i] : pred[i];
        delta[i] = (t > 0) ? 1.0f : -1.0f;
    }
}