#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID
    int stride = blockDim.x * gridDim.x; // Calculate stride based on total threads

    for (int idx = i; idx < N; idx += stride) // Loop to handle larger-than-thread-count data
    {
        Y[idx * INCY] *= X[idx * INCX];
    }
}