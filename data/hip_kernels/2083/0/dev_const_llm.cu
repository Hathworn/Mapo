#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dev_const(float *px, float k) {
    // Use the total number of threads and stride to handle larger data sizes
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = idx; i < stride; i += stride) {
        px[i] = k;  // Efficient memory access using for loop
    }
}