#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // Handle the data at this index
    if (tid < N) // Ensure within bounds before loop
    {
        c[tid] = a[tid] + b[tid];
    }
    tid += blockDim.x * gridDim.x;
    while (tid < N)
    {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}