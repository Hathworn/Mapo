#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Use block and thread indexing for better scalability
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) { // Ensure i is within bounds
        c[i] = a[i] + b[i];
    }
}