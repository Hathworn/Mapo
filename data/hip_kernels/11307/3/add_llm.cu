#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use a more efficient thread indexing strategy
__global__ void add(int *a, int *b, int *c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // calculate global thread ID
    if (tid < N)
        c[tid] = a[tid] + b[tid]; // perform addition only if within bounds
}