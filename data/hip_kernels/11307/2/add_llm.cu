#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c)
{
    // Use shared memory to reduce global memory access latency
    __shared__ int temp;
    
    if (threadIdx.x == 0) {
        temp = a + b; // Compute the sum using a single thread
    }
    
    __syncthreads();
    
    if (threadIdx.x == 0) {
        *c = temp; // Store the result in the global memory
    }
}