#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(float *a, float *b, float *c)
{
    extern __shared__ float cache[]; // Dynamically allocated shared memory
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0f;
    int stride = blockDim.x * gridDim.x;

    // Unrolling the loop for better performance
    for (int i = tid; i < N; i += stride)
    {
        temp += a[i] * b[i];
    }
    cache[cacheIndex] = temp;

    __syncthreads();

    // Optimized reduction using warp shuffle for first 32 threads
    for (int d = warpSize / 2; d > 0; d /= 2)
    {
        if (cacheIndex < d)
        {
            cache[cacheIndex] += __shfl_down_sync(0xFFFFFFFF, cache[cacheIndex], d);
        }
    }

    if (cacheIndex < warpSize)
    {
        // Additional reduction for multiple warps
        volatile float *v_cache = cache;
        if (cacheIndex < 16) v_cache[cacheIndex] += v_cache[cacheIndex + 16];
        if (cacheIndex < 8)  v_cache[cacheIndex] += v_cache[cacheIndex + 8];
        if (cacheIndex < 4)  v_cache[cacheIndex] += v_cache[cacheIndex + 4];
        if (cacheIndex < 2)  v_cache[cacheIndex] += v_cache[cacheIndex + 2];
        if (cacheIndex < 1)  v_cache[cacheIndex] += v_cache[cacheIndex + 1];
    }

    if (cacheIndex == 0)
    {
        c[blockIdx.x] = cache[0];
    }
}