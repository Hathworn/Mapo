#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(unsigned char *ptr, int ticks) {
    // Calculate global thread position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Avoid unnecessary memory operations if out of bounds
    if (x >= DIM || y >= DIM) return;

    int offset = x + y * blockDim.x * gridDim.x;
    
    // Pre-compute constant values to reduce redundant calculations
    float fx = x - DIM * 0.5f;
    float fy = y - DIM * 0.5f;
    float distance = sqrtf(fx * fx + fy * fy);
    float cosine_value = cosf(distance * 0.1f - ticks * 0.142857f); // Precompute 1/7

    // Compute pixel color
    unsigned char grey = (unsigned char)(128.0f + 127.0f * cosine_value / (distance * 0.1f + 1.0f));
    ptr[offset * 4 + 0] = grey;
    ptr[offset * 4 + 1] = grey;
    ptr[offset * 4 + 2] = grey;
    ptr[offset * 4 + 3] = 255;
}