#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel( uchar4 *ptr, int ticks ) {
    // Precompute constants outside the loop
    int half_dim = DIM / 2;
    float ticks_factor = ticks / 7.0f;

    // Map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // Calculate the value at that position
    float fx = x - half_dim;
    float fy = y - half_dim;
    float d = sqrtf(fx * fx + fy * fy);
    float denom = d / 10.0f + 1.0f;
    // Perform the cosine operation
    float cos_val = cosf(d / 10.0f - ticks_factor);
    unsigned char grey = (unsigned char)(128.0f + 127.0f * (cos_val / denom));
    
    // Store computed grey level in uchar4
    ptr[offset].x = grey;
    ptr[offset].y = grey;
    ptr[offset].z = grey;
    ptr[offset].w = 255;
}