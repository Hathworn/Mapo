#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Use shared memory for faster access
    __shared__ int shared_a[1024];
    __shared__ int shared_b[1024];

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // Load elements into shared memory
    shared_a[threadIdx.x] = a[i];
    shared_b[threadIdx.x] = b[i];
    __syncthreads();

    // Compute the result using shared memory
    c[i] = shared_a[threadIdx.x] + shared_b[threadIdx.x];
}