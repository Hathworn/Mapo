#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    // Calculate pixel coordinates
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return; // Early return if out-of-bounds

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate position with offset and scaling
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // Decompose into integer coordinates and fractional offsets
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    int tx = (int) px;
    int ty = (int) py;

    float value = src[image_row_offset + j];

    // Macro to avoid repetitive bounds checking and operations
    #define UPDATE_PIXEL(off_tx, off_ty, weight_factor) \
    { \
        int final_tx = tx + (off_tx); \
        int final_ty = ty + (off_ty); \
        if (final_tx >= 0 && final_tx < w && final_ty >= 0 && final_ty < h) \
        { \
            float weight = weight_factor; \
            _atomicAdd(dst + final_ty * image_stride + final_tx, value * weight); \
            _atomicAdd(normalization_factor + final_ty * image_stride + final_tx, weight); \
        } \
    }

    // Update the pixels at the four corners with appropriate weights
    UPDATE_PIXEL(0, 0, dx * dy);              // Bottom-right
    UPDATE_PIXEL(-1, 0, (1.0f - dx) * dy);    // Bottom-left
    UPDATE_PIXEL(-1, -1, (1.0f - dx) * (1.0f - dy)); // Upper-left
    UPDATE_PIXEL(0, -1, dx * (1.0f - dy));    // Upper-right

    #undef UPDATE_PIXEL
}