#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global thread index for 2D grid
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if within image bounds
    if (i < h && j < w) {
        // Linearize 2D coordinate to 1D for accessing arrays
        const int pos = i * s + j;

        // Avoid divide by zero
        float scale = normalization_factor[pos];
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

        // Normalize pixel value
        image[pos] *= invScale;
    }
}