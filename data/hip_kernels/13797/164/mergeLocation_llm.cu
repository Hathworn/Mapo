#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale) {
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by limiting memory access and arithmetic operations
    if (ptidx < npoints) {
        short2 loc = __ldg(&loc_[ptidx]); // Use __ldg to cache global memory

        float scaled_x = loc.x * scale;
        float scaled_y = loc.y * scale;

        x[ptidx] = scaled_x;
        y[ptidx] = scaled_y;
    }
}