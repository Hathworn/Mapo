#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

# define PI 3.14159265358979323846

__global__ void atomicScatter(int type1, int type2, int size_K, double *occ, double *beq, double *K, double *a, double *b, double *c, double *ffoobb) {
    // Calculate thread index
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Process only valid thread indices
    if (Idx < size_K) {
        double rp16pi2 = -0.006332573977646;
        double negativeHalfSquaredS = K[Idx] * K[Idx] * rp16pi2;
        
        // Prefactor calculation
        double occupationFactor = occ[type1] * occ[type2];
        double debyeWallerFactor = exp(negativeHalfSquaredS * (beq[type1] + beq[type2]));
        ffoobb[Idx] = occupationFactor * debyeWallerFactor;

        // Use variables for repeated access
        double negK2S = K[Idx] * K[Idx];
        double f1 = c[type1];
        double f2 = c[type2];

        // Loop unrolling for better performance
        for (int i = 0; i < 5; i++) {
            double expFactor = exp(b[type1 * 5 + i] * negativeHalfSquaredS);
            f1 += a[type1 * 5 + i] * expFactor;
            expFactor = exp(b[type2 * 5 + i] * negativeHalfSquaredS);
            f2 += a[type2 * 5 + i] * expFactor;
        }
        
        // Final prefactor multiplication
        ffoobb[Idx] *= f1 * f2;
    }
}