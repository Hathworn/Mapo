#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel function by utilizing shared memory
__global__ void fillKernel(int *a, int n) {
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Check boundary condition only once
    if (tid < n) {
        a[tid] = tid;
    }
}