#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot(float * val, int *row_ind, int *col_ind, int nnz, float * ret, float * w) {
    // Calculate unique global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride loop to handle larger data sets more efficiently
    for (int index = tid; index < nnz; index += blockDim.x * gridDim.x) {
        int r = row_ind[index];
        int c = col_ind[index];
        float v = val[index];
        atomicAdd(&ret[r], v * w[c]);
    }
}