#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void grad(float *val, int *row_ind, int *col_ind, float *mat_err, int nnz, float *act, float *label, float *w, float learning_rate) {
    // Calculate the unique index for each thread
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread is within bounds
    if (tid < nnz) {
        // Cache accesses to global memory
        int r = row_ind[tid];
        int c = col_ind[tid];
        float v = val[tid];
        
        // Compute the error and update the weight
        float difference = label[r] - act[r];
        mat_err[tid] = abs(difference);
        float err = v * difference;
        atomicAdd(&w[c], learning_rate * err);
    }
}