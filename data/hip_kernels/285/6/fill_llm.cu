#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill(float * w, float val, int size) {
    // Use a stride equal to the total number of threads
    for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < size; tid += blockDim.x * gridDim.x) {
        w[tid] = val; // Assign value
    }
}