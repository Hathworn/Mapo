#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_sigmoid(float * d, int num) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process elements if thread ID is within bounds
    if (tid < num) {
        float val = d[tid];
        
        // Use ternary operators for concise conditional checks
        d[tid] = (val > 10.0) ? 1.0 : ((val < -10.0) ? 0.0 : 1.0 / (1.0 + expf(-val)));
    }
}