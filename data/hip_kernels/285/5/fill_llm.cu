#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill(float * w, float val, int size) {
    // Calculate global thread ID
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll loop for potential performance gain
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        w[i] = val;
    }
}