#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*****************************************************************************/
/* Вычислить активационную функцию y(x) = 2x / (1 + abs(x)). */
/*****************************************************************************/
//inline REAL OpenNNL::activation(REAL x, TActivationKind kind)
__global__ void initializeRandomGenerator ( hiprandState * state, unsigned long seed, int count )
{
int idx = blockIdx.x * blockDim.x + threadIdx.x;

if(idx < count)
hiprand_init ( seed, idx, 0, &state[idx] );
}