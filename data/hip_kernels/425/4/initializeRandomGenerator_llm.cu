#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use a more descriptive name for the kernel
__global__ void initializeRandomGenerator(hiprandState *state, unsigned long seed, int count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check index before processing
    if (idx < count) {
        // Initialize RNG state for each thread
        hiprand_init(seed, idx, 0, &state[idx]);
    }
}