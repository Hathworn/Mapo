#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void InitializeMatrix_kernel(int8_t *matrix, int ldm, int rows, int columns) {
    // Optimize index calculation for improved readability and remove redundant checks
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < rows && j < columns) {
        int offset = i + j * ldm;
        matrix[offset] = 0;

        // Combine conditions since both set matrix[offset] to 0x0
        if ((i >= rows - 2 && j < 1) || (i < 1 && j >= columns - 2)) {
            matrix[offset] = 0x0;
        }
    }
}