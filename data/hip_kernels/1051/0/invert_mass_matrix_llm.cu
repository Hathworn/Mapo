#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel function
__global__ void invert_mass_matrix(double *values, unsigned int size)
{
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;

    // Check if index is within bounds
    if (i >= size) return;

    // Precompute reciprocal and store, avoiding branching
    double val = values[i];
    values[i] = (val > 1e-15) ? (1.0 / val) : 0.0;
}