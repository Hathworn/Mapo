#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

long long remaining_N2(int , int ,long long );
long long remaining_N(int , int ,int );

__global__ void ker2(float * cormat, float * upper, int n1, int n, long long upper_size, int N, int i_so_far, long long M1)
{
    // Calculate the global index using built-in functions for efficiency
    long long idx = blockIdx.x * blockDim.x + threadIdx.x; 
    long i = idx / n;
    long j = idx % n;

    // Reduce condition checks for efficiency
    if (i < n1 && j < n && i < j)
    {
        long long tmp_2 = i * n - i * (i + 1) / 2 + j - i;
        long long indexi = n1 * j + i;
        upper[tmp_2 - 1] = cormat[indexi];
    }
}