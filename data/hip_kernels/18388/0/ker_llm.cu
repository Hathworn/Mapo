#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

long long remaining_N2(int , int ,long long );
long long remaining_N(int , int ,int );

// Optimized kernel function
__global__ void ker(float * cormat, float * upper, int n1, int n)
{
    long idx = blockDim.x * blockIdx.x + threadIdx.x;
    long i = idx % n1;
    long j = idx / n1;
    
    // Check condition where i < j, eliminating unnecessary operations
    if (i < j && j < n)
    {
        long tmp_2 = i * n + j - (i * (i + 1) / 2) - i;
        
        // Access global memory more efficiently
        upper[tmp_2 - 1] = cormat[j * n + i];
    }
}