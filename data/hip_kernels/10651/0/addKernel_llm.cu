#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x; // Compute global index
    c[index] = a[index] + b[index]; // Perform addition based on global index
}