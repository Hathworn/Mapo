#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function for dot product
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numCols, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f; // Local accumulator

    // Loop to calculate partial dot product
    for (; eidx < numElements; eidx += numCols) {
        sum += a[eidx] * b[eidx];
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Unrolling with single warp shuffle reduction
    if (threadIdx.x < 32) {
        for (int offset = 16; offset > 0; offset >>= 1) {
            sum += __shfl_down_sync(0xFFFFFFFF, sum, offset); // Warp shuffle for reduction
        }
        if (threadIdx.x == 0) {
            target[blockIdx.x] = sum; // Store result
        }
    }
}