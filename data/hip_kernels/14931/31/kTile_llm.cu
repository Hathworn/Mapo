#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total number of threads
    const int numThreads = blockDim.x * gridDim.x;

    // Loop through target elements, using step size equal to the number of threads
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;  // Determine y position in target
        const uint x = i % tgtWidth;  // Determine x position in target

        const uint srcY = y % srcHeight;  // Wrap y to source height
        const uint srcX = x % srcWidth;   // Wrap x to source width

        tgt[i] = src[srcY * srcWidth + srcX];  // Map source to target
    }
}