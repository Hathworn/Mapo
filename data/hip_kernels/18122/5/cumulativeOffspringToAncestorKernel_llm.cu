#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cumulativeOffspringToAncestorKernel(const int* cumulativeOffspring, int* ancestor, int numParticles) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= numParticles) return; // Simplified boundary check

    // Use global memory efficiently by removing redundant access
    int start = (idx == 0) ? 0 : cumulativeOffspring[idx - 1];
    int numCurrentOffspring = cumulativeOffspring[idx] - start;

    for (int j = 0; j < numCurrentOffspring; j++) {
        ancestor[start + j] = idx;
    }
}