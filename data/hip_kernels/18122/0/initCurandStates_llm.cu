#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* File smc_kernels.cu contains definitions of kernels used by SMC.
*/

// Check if using NVCC
#ifdef __NVCC__
#endif

// Kernel to initialize CURAND states
__global__ void initCurandStates(hiprandState* randStates, int numThreads, int seed) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numThreads) return;

    // Use local variable to optimize memory access and avoid unnecessary global accesses
    hiprandState localRandState;
    hiprand_init(1234 + clock64(), seed * numThreads + i, 0, &localRandState);
    randStates[i] = localRandState;
}