#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adadeltaKernel (int numberIterations, int* parameterIndices, int* counts, int dimension, float* parameters, float* gradient, float decay, float oneMinusDecay, float epsilon, float* gradientAccumulation, float* updateAccumulation) {

    int updateIndex = blockIdx.x;
    int parameterIndex = parameterIndices[updateIndex];
    int count = counts[updateIndex];

    if(parameterIndex != -1 && count > 0) {

        float scalingFactor = 1.0f / (float)count;

        // Calculate start entry indices
        int startEntryIndex = (blockIdx.y * blockDim.x + threadIdx.x) * numberIterations;
        int firstParameterEntryIndex = parameterIndex * dimension;
        int startParameterEntryIndex = firstParameterEntryIndex + startEntryIndex;
        int startGradientEntryIndex = updateIndex * dimension + startEntryIndex;

        // Determine exclusive end index, avoiding read out of bounds
        int exclusiveEndParameterEntryIndex = min(startParameterEntryIndex + numberIterations, firstParameterEntryIndex + dimension);

        // Loop using appropriate indices to update parameters
        for(int i = startParameterEntryIndex, j = startGradientEntryIndex; i < exclusiveEndParameterEntryIndex; ++i, ++j) {
            float scaledDerivative = scalingFactor * gradient[j];

            // Update gradient accumulation
            float newGradientAccumulation = decay * gradientAccumulation[i] + oneMinusDecay * (scaledDerivative * scaledDerivative);
            gradientAccumulation[i] = newGradientAccumulation;

            // Compute RMS values
            float rootMeanSquaredOfDerivatives = sqrtf(newGradientAccumulation + epsilon);
            float pastUpdateAccumulation = updateAccumulation[i];
            float rootMeanSquaredOfPastUpdates = sqrtf(pastUpdateAccumulation + epsilon);

            // Compute learning rate and update value
            float learningRate = rootMeanSquaredOfPastUpdates / rootMeanSquaredOfDerivatives;
            float update = -learningRate * scaledDerivative;

            // Update the accumulation and parameter
            updateAccumulation[i] = decay * pastUpdateAccumulation + oneMinusDecay * (update * update);
            parameters[i] += update;
        }
    }
}