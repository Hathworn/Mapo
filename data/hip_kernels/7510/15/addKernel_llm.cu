#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(float* A, float* B, float* C, int numberIterations, int size) {

    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int start = idx * numberIterations;
    
    // Unroll the loop to improve performance
    #pragma unroll
    for (int i = 0; i < numberIterations && start + i < size; ++i) {
        int entryIndex = start + i;
        C[entryIndex] = A[entryIndex] + B[entryIndex];
    }
}