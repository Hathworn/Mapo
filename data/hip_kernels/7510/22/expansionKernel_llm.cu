#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expansionKernel( int batchSize, float* input, int* inputLengths, int numberRows, int numberInputEntries, int numberFilterRowPositions, int filterHeight, int filterWidth, int filterLength, int numberResultEntries, float* result, int* resultLengths) {

    int indexInstance = blockIdx.x;
    int indexConvolution = blockIdx.y;
    int indexConvolutionEntry = threadIdx.x;

    int firstColumnOfConvolution = indexConvolution / numberFilterRowPositions;
    int firstRowOfConvolution = indexConvolution % numberFilterRowPositions;

    int relativeIndexColumn = indexConvolutionEntry / filterHeight;
    int relativeIndexRow = indexConvolutionEntry % filterHeight;

    int indexColumn = firstColumnOfConvolution + relativeIndexColumn;
    int indexRow = firstRowOfConvolution + relativeIndexRow;

    int indexEntryWithinResult = indexInstance * numberResultEntries + indexConvolution * filterLength + indexConvolutionEntry;

    if(indexInstance < batchSize) {
        int inputLength = inputLengths[indexInstance];

        // Calculating number of convolutions inline and directly using it
        int numberConvolutions = (inputLength - filterWidth + 1) * numberFilterRowPositions;

        if(indexConvolution < numberConvolutions) {
            result[indexEntryWithinResult] = input[indexInstance * numberInputEntries + indexColumn * numberRows + indexRow];
        } else {
            result[indexEntryWithinResult] = 0.0f; // Using float literal for better type consistency
        }

        // Single thread responsible for setting result lengths
        if(indexConvolution == 0 && indexConvolutionEntry == 0) {
            resultLengths[indexInstance] = numberConvolutions;
        }
    } else {
        result[indexEntryWithinResult] = nanf("NaN"); // NaN assignment unchanged
        // Avoid setting resultLengths for out of range indexInstance
    }
}