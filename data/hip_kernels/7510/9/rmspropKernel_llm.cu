#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rmspropKernel(int numberIterations, int* parameterIndices, int* counts, int dimension, float* parameters, float* gradient, float learningRate, float decay, float oneMinusDecay, float epsilon, float* accumulation) {

  // Optimize thread index calculation using built-in variables
  int updateIndex = blockIdx.x;
  int parameterIndex = parameterIndices[updateIndex];
  int count = counts[updateIndex];

  if (parameterIndex != -1 && count > 0) {

    // Pre-calculate scaling factor outside loop
    float scalingFactor = 1.0f / (float)count;

    // Optimize entry index calculation for better readability
    int startEntryIndex = threadIdx.x + blockIdx.y * blockDim.x;
    int firstParameterEntryIndex = parameterIndex * dimension;
    int startParameterEntryIndex = firstParameterEntryIndex + startEntryIndex * numberIterations;
    int startGradientEntryIndex = updateIndex * dimension + startEntryIndex * numberIterations;

    // Optimize variable naming and indexing for clarity and performance
    int exclusiveEndParameterEntryIndex = min(startParameterEntryIndex + numberIterations, firstParameterEntryIndex + dimension);

    for (int i = 0; i < numberIterations && startParameterEntryIndex + i < exclusiveEndParameterEntryIndex; i++) {

      float scaledDerivative = scalingFactor * gradient[startGradientEntryIndex + i];

      float updatedAccumulation = decay * accumulation[startParameterEntryIndex + i] + oneMinusDecay * (scaledDerivative * scaledDerivative);
      accumulation[startParameterEntryIndex + i] = updatedAccumulation;

      float adaptiveLearningRate = learningRate / sqrtf(updatedAccumulation + epsilon);
      float update = -adaptiveLearningRate * scaledDerivative;

      parameters[startParameterEntryIndex + i] += update;
    }
  }
}