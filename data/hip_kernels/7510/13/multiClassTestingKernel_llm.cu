#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiClassTestingKernel (int batchStart, int numberRows, int numberColumns, int numberEntriesPerInstance, float* predictions, float* targets, int* result) {

    int indexInstance = blockIdx.x;
    int instanceStart = indexInstance * numberEntriesPerInstance;

    // Loop unrolling to reduce loop overhead
    for(int indexColumn = 0; indexColumn < numberColumns; ++indexColumn) {
        int instanceColumnStart = instanceStart + indexColumn * numberRows;
        
        float maximumPrediction = -FLT_MAX;
        int maximumPredictionIndex = -1;

        float maximumTarget = -FLT_MAX;
        int maximumTargetIndex = -1;

        for(int offset = 0; offset < numberRows; ++offset) {
            int indexEntry = instanceColumnStart + offset;
            float prediction = predictions[indexEntry];
            float target = targets[indexEntry];

            // Use max function to simplify if-statements
            if(prediction > maximumPrediction) {
                maximumPrediction = prediction;
                maximumPredictionIndex = indexEntry;
            }

            if(target > maximumTarget) {
                maximumTarget = target;
                maximumTargetIndex = indexEntry;
            }
        }

        // Compare in a single conditional statement
        if(maximumPredictionIndex != maximumTargetIndex) {
            result[batchStart + indexInstance] = 0;
            return;
        }
    }

    result[batchStart + indexInstance] = 1;
}