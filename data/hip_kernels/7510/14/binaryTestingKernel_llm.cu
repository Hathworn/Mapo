#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binaryTestingKernel(int batchStart, int length, float* predictions, float* targets, int* result) {
    int withinBatch = blockIdx.x;
    int threadId = threadIdx.x; // Get thread index
    int instanceStart = batchStart + withinBatch * length + threadId; // Offset by thread index
    int stride = blockDim.x; // Calculate stride

    // Use stride to handle multiple elements per thread
    for (int indexEntry = instanceStart; indexEntry < batchStart + (withinBatch + 1) * length; indexEntry += stride) {
        float prediction = predictions[indexEntry];
        float target = targets[indexEntry];

        result[indexEntry] = (prediction < 0.5 && target == 0.0) || (prediction >= 0.5 && target == 1.0);
    }
}