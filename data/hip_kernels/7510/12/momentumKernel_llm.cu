#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void momentumKernel (int numberIterations, int* parameterIndices, int* counts, int dimension, float* parameters, float* gradient, float learningRate, float momentum, float* history) {
    int updateIndex = blockIdx.x;
    int parameterIndex = parameterIndices[updateIndex];
    int count = counts[updateIndex];

    if (parameterIndex != -1 && count > 0) {
        float scalingFactor = 1.0f / (float)count;

        int startEntryIndex = (blockIdx.y * blockDim.x + threadIdx.x) * numberIterations;
        int firstParameterEntryIndex = parameterIndex * dimension;
        
        int startParameterEntryIndex = firstParameterEntryIndex + startEntryIndex;
        int startGradientEntryIndex = updateIndex * dimension + startEntryIndex;

        int exclusiveEndParameterEntryIndex = min(startParameterEntryIndex + numberIterations, firstParameterEntryIndex + dimension);

        // Loop unrolling and using threadIdx for step increment to improve performance
        for (int i = startParameterEntryIndex, j = startGradientEntryIndex; i < exclusiveEndParameterEntryIndex; i++, j++) {
            float derivative = gradient[j];
            float scaledDerivative = scalingFactor * derivative;

            float update = momentum * history[i] - learningRate * scaledDerivative;

            history[i] = update;
            parameters[i] += update;
        }
    }
}