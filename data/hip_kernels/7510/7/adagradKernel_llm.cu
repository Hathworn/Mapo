#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adagradKernel (int numberIterations, int* parameterIndices, int* counts, int dimension, float* parameters, float* gradient, float learningRate, float* history, float epsilon) {

    // Precompute common values
    int updateIndex = blockIdx.x;
    int parameterIndex = parameterIndices[updateIndex];
    if (parameterIndex == -1) return;  // Optimize conditional check

    int count = counts[updateIndex];
    if (count <= 0) return;  // Optimize conditional check

    float scalingFactor = 1.0f / (float)count;
    int firstParameterEntryIndex = parameterIndex * dimension;
    int startEntryIndex = (blockIdx.y * blockDim.x + threadIdx.x) * numberIterations;

    int startParameterEntryIndex = firstParameterEntryIndex + startEntryIndex;
    int startGradientEntryIndex = updateIndex * dimension + startEntryIndex;
    int exclusiveEndParameterEntryIndex = min(startParameterEntryIndex + numberIterations, firstParameterEntryIndex + dimension);

    // Use local variables to avoid repeated memory accesses
    int parameterEntryIndex = startParameterEntryIndex;
    int gradientEntryIndex = startGradientEntryIndex;

    while (parameterEntryIndex < exclusiveEndParameterEntryIndex) {
        float scaledDerivative = scalingFactor * gradient[gradientEntryIndex];
        float updatedHistory = history[parameterEntryIndex] + scaledDerivative * scaledDerivative;
        history[parameterEntryIndex] = updatedHistory;

        float adaptedLearningRate = learningRate / (sqrtf(updatedHistory) + epsilon);
        float update = adaptedLearningRate * scaledDerivative;  // Avoid redundant scaling multiplication
        parameters[parameterEntryIndex] -= update;

        // Increment indices efficiently
        parameterEntryIndex++;
        gradientEntryIndex++;
    }
}