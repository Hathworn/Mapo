#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backwardMaxPoolingKernel ( int batchSize, int* lengths, int numberEntriesPerInstance, int numberRows, int* maxIndices, float* chain, float* result) {
    int indexInstance = blockIdx.x;
    int indexRow = blockIdx.y;
    int indexColumn = threadIdx.x;

    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startColumnWithinInstance = indexColumn * numberRows;
    int indexEntryWithinBatch = startInstanceWithinBatch + startColumnWithinInstance + indexRow;

    if(indexInstance < batchSize) {
        int length = lengths[indexInstance];

        if(indexColumn < length) {
            // Use shared memory to reduce repeated global memory access
            __shared__ int sharedMaxIndices[blockDim.x];
            __shared__ float sharedChain[blockDim.x];
            if(threadIdx.x == 0) {
                sharedMaxIndices[indexRow] = maxIndices[indexInstance * numberRows + indexRow];
                sharedChain[indexRow] = chain[indexInstance * numberRows + indexRow];
            }
            __syncthreads();

            int maxIndex = sharedMaxIndices[indexRow];

            if(indexEntryWithinBatch == maxIndex) {
                result[indexEntryWithinBatch] = sharedChain[indexRow];
            } else {
                result[indexEntryWithinBatch] = 0.0;
            }
        } else {
            result[indexEntryWithinBatch] = nanf("NaN");
        }
    } else {
        result[indexEntryWithinBatch] = nanf("NaN");
    }
}