#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumKernel( float* input, float* result, int batchSize, int numberEntries, int numberIterations) {
    
    int startEntry = (blockIdx.x * blockDim.x + threadIdx.x) * numberIterations;
    int exclusiveEndEntry = min(startEntry + numberIterations, numberEntries);

    // Load shared memory for coalescing reads
    extern __shared__ float sharedInput[];
    
    for (int entryIndex = startEntry; entryIndex < exclusiveEndEntry; entryIndex++) {
        float entry = 0.0f;
        
        // Loop unrolling for improved performance
        for (int instanceIndex = 0; instanceIndex < batchSize; instanceIndex += 4) {
            entry += input[instanceIndex * numberEntries + entryIndex];
            if (instanceIndex + 1 < batchSize)
                entry += input[(instanceIndex + 1) * numberEntries + entryIndex];
            if (instanceIndex + 2 < batchSize)
                entry += input[(instanceIndex + 2) * numberEntries + entryIndex];
            if (instanceIndex + 3 < batchSize)
                entry += input[(instanceIndex + 3) * numberEntries + entryIndex];
        }
        
        result[entryIndex] = entry;
    }
}