#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void groupSumKernel( float* input, int* firstOccurrences, int* otherOccurrences, int* otherOccurrencePositions, int dimensions, int numberIterations) {

    int duplicateIndex = blockIdx.x;
    int firstOccurrence = firstOccurrences[duplicateIndex];
    int startWithinParameter = threadIdx.x * numberIterations;
    int startFirstOccurrenceWithinBatch = firstOccurrence * dimensions;
    int startFirstOccurrenceEntryIndex = startFirstOccurrenceWithinBatch + startWithinParameter;
    int exclusiveEndFirstOccurrenceEntryIndex = min(startFirstOccurrenceEntryIndex + numberIterations, startFirstOccurrenceWithinBatch + dimensions);
    int startOtherOccurrencePosition = otherOccurrencePositions[duplicateIndex];
    int exclusiveEndOtherOccurrencePosition = otherOccurrencePositions[duplicateIndex + 1];

    for (int otherOccurrencePosition = startOtherOccurrencePosition; otherOccurrencePosition < exclusiveEndOtherOccurrencePosition; otherOccurrencePosition++) {
        int otherOccurrence = otherOccurrences[otherOccurrencePosition];
        int otherOccurrenceEntryIndex = otherOccurrence * dimensions + startWithinParameter;

        // Use batched processing to minimize loop overhead
        for (int i = startFirstOccurrenceEntryIndex; i < exclusiveEndFirstOccurrenceEntryIndex; ++i) {
            input[i] += __ldg(&input[otherOccurrenceEntryIndex + i - startFirstOccurrenceEntryIndex]);
            input[otherOccurrenceEntryIndex + i - startFirstOccurrenceEntryIndex] = nanf("NaN");
        }
    }
}