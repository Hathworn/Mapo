#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nesterovKernel(int numberIterations, int* parameterIndices, int* counts, int dimension, float* parameters, float* gradient, float learningRate, float momentum, float* history, float* backup) {

    int updateIndex = blockIdx.x;
    int parameterIndex = parameterIndices[updateIndex];
    int count = counts[updateIndex];

    if (parameterIndex != -1 && count > 0) {
        float scalingFactor = 1.0f / (float)count;
        int startEntryIndex = (blockIdx.y * blockDim.x + threadIdx.x) * numberIterations;

        int firstParameterEntryIndex = parameterIndex * dimension;
        int startParameterEntryIndex = firstParameterEntryIndex + startEntryIndex;
        int exclusiveEndParameterEntryIndex = min(startParameterEntryIndex + numberIterations, firstParameterEntryIndex + dimension);

        int startGradientEntryIndex = updateIndex * dimension + startEntryIndex;

        int parameterEntryIndex = startParameterEntryIndex;
        int gradientEntryIndex = startGradientEntryIndex;

        // Loop optimization for parallel computation
        #pragma unroll
        for (int i = startParameterEntryIndex; i < exclusiveEndParameterEntryIndex; i++, gradientEntryIndex++) {
            float entryBackup = history[i];
            backup[i] = entryBackup;

            float scaledDerivative = scalingFactor * gradient[gradientEntryIndex];
            float entryUpdate = momentum * history[i] - learningRate * scaledDerivative;
            history[i] = entryUpdate;

            float removedPreviousLookAhead = parameters[i] - momentum * entryBackup;
            parameters[i] = removedPreviousLookAhead + (1.0f + momentum) * entryUpdate;
        }
    }
}