#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stochasticGradientDescentKernel(int numberIterations, int* parameterIndices, int* counts, int dimension, float* parameters, float* gradient, float learningRate) {

    int updateIndex = blockIdx.x;
    int parameterIndex = parameterIndices[updateIndex];
    int count = counts[updateIndex];

    // Check if parameterIndex is valid and count is greater than zero
    if (parameterIndex != -1 && count > 0) {
        // Pre-compute scaling factor
        float scalingFactor = 1.0f / (float)count;

        // Calculate starting indexes
        int startEntryIndex = (blockIdx.y * blockDim.x + threadIdx.x) * numberIterations;
        int firstParameterEntryIndex = parameterIndex * dimension;
        int startParameterEntryIndex = firstParameterEntryIndex + startEntryIndex;
        int startGradientEntryIndex = updateIndex * dimension + startEntryIndex;

        // Use min to ensure we do not exceed the dimension
        int exclusiveEndParameterEntryIndex = min(startParameterEntryIndex + numberIterations, firstParameterEntryIndex + dimension);

        // Loop through each entry while performing the update
        for (int i = startParameterEntryIndex, j = startGradientEntryIndex; i < exclusiveEndParameterEntryIndex; ++i, ++j) {
            float scaledDerivative = scalingFactor * gradient[j];
            parameters[i] -= learningRate * scaledDerivative;
        }
    }
}