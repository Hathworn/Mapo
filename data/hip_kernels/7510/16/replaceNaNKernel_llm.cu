#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void replaceNaNKernel(int numberEntriesPerInstance, int numberIterations, float* source, float* destination) {
    // Calculate the global thread index
    int globalIndex = blockIdx.x * blockDim.x * numberIterations + blockIdx.y * blockDim.x + threadIdx.x;

    // Iterate over elements with a stride equal to the total number of threads
    for (int idx = globalIndex; idx < numberEntriesPerInstance * gridDim.x; idx += blockDim.x * gridDim.y) {
        float currentValue = source[idx];
        destination[idx] = isnan(currentValue) ? 0.0f : currentValue;
    }
}