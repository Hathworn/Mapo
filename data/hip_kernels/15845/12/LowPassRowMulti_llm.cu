#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LowPassRowMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    __shared__ float data[CONVROW_W + 2*RADIUS]; // Shared memory for current block
    const int tx = threadIdx.x;
    const int block = blockIdx.x / (NUM_SCALES + 3);
    const int scale = blockIdx.x - (NUM_SCALES + 3) * block;
    const int xout = block * CONVROW_W + tx;
    const int loadPos = xout - RADIUS;
    const int yptr = blockIdx.y * pitch;
    const int writePos = yptr + height * pitch * scale + xout;
    float *kernel = d_Kernel + scale * 16;

    // Using clamp to load data safely
    int safeLoadPosLeft = max(loadPos, 0);
    int safeLoadPosRight = min(loadPos, width - 1);
    data[tx] = d_Data[yptr + (loadPos < 0 || loadPos >= width ? safeLoadPosRight : loadPos)];
    __syncthreads();

    // Ensure valid position before writing result
    if (xout < width && tx < CONVROW_W) {
        float sum = 0.0f;
        for (int i = 0; i < 5; ++i) {
            sum += (data[tx + i] + data[tx + 8 - i]) * kernel[i];
        }
        d_Result[writePos] = sum;
    }
    __syncthreads();
}