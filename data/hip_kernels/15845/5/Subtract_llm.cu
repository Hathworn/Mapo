#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    const int x = blockIdx.x * SUBTRACT_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACT_H + threadIdx.y;
  
    // Optimize by eliminating unneeded synchronization
    if (x < width && y < height) {
        int p = y * pitch + x;
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // __syncthreads() not needed here as there is no shared memory
}