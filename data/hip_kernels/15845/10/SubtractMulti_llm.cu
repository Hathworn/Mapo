#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim.x for better flexibility
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Use blockDim.y for better flexibility

    if (x < width && y < height) // Check bounds before calculation
    {
        int sz = height * pitch;
        int p = threadIdx.z * sz + y * pitch + x; // Rearrange calculations for clarity
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
    // __syncthreads() removed as it's unnecessary without shared memory
}