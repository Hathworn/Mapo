#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    int p = y * pitch + x;

    // Load initial values to shared memory
    float initial_val;
    if (x < width) {
        initial_val = d_Data[p];
    } else {
        initial_val = d_Data[y * pitch + (blockIdx.x * 128)];
    }
    minvals[tx] = initial_val;
    maxvals[tx] = initial_val;

    // Loop over rows and perform min/max reduction
    for (int ty = 1; ty < 16; ty++) {
        p += pitch;
        if (x < width) {
            float val = d_Data[p];
            if (val < minvals[tx])
                minvals[tx] = val;
            if (val > maxvals[tx])
                maxvals[tx] = val;
        }
    }
    __syncthreads();

    // Parallel reduction within block
    for (int stride = b / 2; stride > 0; stride >>= 1) {
        if (tx < stride) {
            if (minvals[tx + stride] < minvals[tx])
                minvals[tx] = minvals[tx + stride];
            if (maxvals[tx + stride] > maxvals[tx])
                maxvals[tx] = maxvals[tx + stride];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}