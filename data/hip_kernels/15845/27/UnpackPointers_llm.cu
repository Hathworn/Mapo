#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;
    int width = w / 16;
    int height = h / 32;

    // Loop tiling to improve memory access patterns
    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            unsigned int val = minmax[y * w + x * 16 + tx];
            if (val) {
                int baseIdx = y * 32 * w + x * 16 + tx;
                
                #pragma unroll // Unroll loop for performance
                for (int k = 0; k < 32; ++k) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = baseIdx + k * w;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}