#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Calculate x and y using block and thread indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Compute linear index
    int p = y * width + x;
    
    // Check within bounds and perform multiply-add operation
    if (x < width && y < height) {
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
    __syncthreads();
}