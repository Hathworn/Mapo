#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;
    int total_w = h * w; // Calculate total width
    int batchSize = h / (blockDim.x * 2); // Calculate batch size to process more elements
    for (int i=tx; i<total_w; i += blockDim.x * batchSize) {
        int y = i / w;
        int x = i % w;
        unsigned int val = minmax[y*w + x];
        
        // Optimize by unrolling the loop
        if (val) {
            #pragma unroll
            for (int k=0; k<32; k++) {
                if (val & 0x1 && numPts < maxPts) {
                    ptrs[16*numPts + tx] = (y*32 + k)*w + x + tx;
                    numPts++;
                }
                val >>= 1;
            }
        }
    }
}
```
