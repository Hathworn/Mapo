#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = gridDim.x * blockDim.x; // Use direct variable for stride
    
    for (unsigned int i = idx; i < numEls; i += stride) { // Use 'stride' for clarity and potential performance gain
        dest[i] = a[i] + b[i];
    }
}