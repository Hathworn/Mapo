#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Edge struct.
typedef struct {
    int* startPoints;
    int* endPoints;
    double* weights;
} Edge;

// Optimized kernel to update the queue
__global__ void updateQueueKernel(int *queueu, int *queueSize, const int *startPoints, const int *endPoints, const int *visitedArray, const int *currentVertex) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;  // Global thread index
    if (index < *queueSize) {  // Ensure within bounds
        if (startPoints[index] == *currentVertex && visitedArray[endPoints[index]] == 0) {
            int oldValue = atomicAdd(queueSize, 1);  // Atomically increment queueSize
            queueu[oldValue] = index;  // Assign thread index that meets condition
        }
    }
}