#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

// Edge struct.
typedef struct {
    int* startPoints;
    int* endPoints;
    double* weights;
} Edge;

// Optimized kernel using parallelization to improve performance
__global__ void processQueueKernel(int *parentArray, double *resultWeightArray, const int* queue, const int *startPoints, const int *endPoints, const double *weightArray, int queueSize) {
    int threadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread does not exceed queue size
    if (threadIndex < queueSize) {
        int elementIndex = queue[threadIndex];
        int startNode = startPoints[elementIndex];
        int endNode = endPoints[elementIndex];
        double edgeWeight = weightArray[elementIndex];
        double nodeWeight = resultWeightArray[startNode];
        
        // Atomic operation for safe concurrent updates
        atomicMin(&resultWeightArray[endNode], nodeWeight + edgeWeight);
        if (nodeWeight + edgeWeight == resultWeightArray[endNode]) {
            parentArray[endNode] = startNode;
        }
    }
}