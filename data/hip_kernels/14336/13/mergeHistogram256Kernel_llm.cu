#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeHistogram256Kernel( uint *d_Histogram, uint *d_PartialHistograms, uint histogramCount )
{
    uint sum = 0;

    // Unroll loop using warp-level primitives
    for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)
    {
        // Efficient memory access for histogram accumulation
        sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM256_BIN_COUNT];
    }

    // Shared memory for intra-block reduction
    __shared__ uint data[MERGE_THREADBLOCK_SIZE];
    data[threadIdx.x] = sum;

    // Reduce within block using warp shuffle
    for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
    {
        __syncthreads();
        if (threadIdx.x < stride)
        {
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
    }

    // Write the block result to global memory
    if (threadIdx.x == 0)
    {
        d_Histogram[blockIdx.x] = data[0];
    }
}