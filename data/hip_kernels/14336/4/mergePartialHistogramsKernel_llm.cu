#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define UMUL(a, b) ( (a) * (b) )
#define UMAD(a, b, c) ( UMUL((a), (b)) + (c) )

typedef unsigned int uint;
typedef unsigned short ushort;
typedef unsigned char uchar;

#define SHARED_MEMORY_SIZE 49152
#define MERGE_THREADBLOCK_SIZE 128

static uint *d_PartialHistograms;

/*
*	Function that maps value to bin in range 0 inclusive to binCOunt exclusive
*/
__global__ void mergePartialHistogramsKernel(uint *d_Histogram, uint *d_PartialHistograms, uint histogramCount, uint binCount)
{
    // Optimize bin iteration using blockDim.x for better parallelism 
    for (uint bin = blockIdx.x * blockDim.x + threadIdx.x; bin < binCount; bin += gridDim.x * blockDim.x)
    {
        uint sum = 0;
        for (uint histogramIndex = threadIdx.y; histogramIndex < histogramCount; histogramIndex += blockDim.y)
        {
            sum += d_PartialHistograms[bin + histogramIndex * binCount];
        }

        __shared__ uint data[MERGE_THREADBLOCK_SIZE];
        data[threadIdx.y * blockDim.x + threadIdx.x] = sum;

        for (uint stride = blockDim.y / 2; stride > 0; stride >>= 1)
        {
            __syncthreads();
            // Simplified reduction within the warp
            if (threadIdx.y < stride)
            {
                data[(threadIdx.y) * blockDim.x + threadIdx.x] += data[(threadIdx.y + stride) * blockDim.x + threadIdx.x];
            }
        }
        
        if (threadIdx.y == 0 && threadIdx.x == 0)
        {
            d_Histogram[bin] = data[0];
        }
    }
}