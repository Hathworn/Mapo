#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define UMUL(a, b) ((a) * (b))
#define UMAD(a, b, c) (UMUL((a), (b)) + (c))

typedef unsigned int uint;
typedef unsigned short ushort;
typedef unsigned char uchar;

#define SHARED_MEMORY_SIZE 49152
#define MERGE_THREADBLOCK_SIZE 128

static uint *d_PartialHistograms;

/*
*	Function that maps value to bin in range 0 inclusive to binCount exclusive
*/
__global__ void clearHistogram(uint *d_Histogram, uint binCount)
{
    // Allocate shared memory for histogram bins
    __shared__ uint s_Histogram[MERGE_THREADBLOCK_SIZE];

    // Initialize shared memory histogram bins to zero
    uint tid = threadIdx.x;
    s_Histogram[tid] = 0;
    __syncthreads();

    // Iterate over histogram bins using shared memory
    for (uint bin = UMAD(blockIdx.x, blockDim.x, threadIdx.x); bin < binCount; bin += UMUL(blockDim.x, gridDim.x)) {
        // Clear histogram using shared memory
        atomicAdd(&s_Histogram[tid % MERGE_THREADBLOCK_SIZE], d_Histogram[bin]);
        d_Histogram[bin] = 0;
    }

    __syncthreads();

    // Write shared memory bins back to global memory
    atomicAdd(&d_Histogram[tid], s_Histogram[tid]);
}