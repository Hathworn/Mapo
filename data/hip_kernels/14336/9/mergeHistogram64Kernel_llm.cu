#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeHistogram64Kernel(uint *d_Histogram, uint *d_PartialHistograms, uint histogramCount) 
{
    __shared__ uint data[MERGE_THREADBLOCK_SIZE];
    uint sum = 0;

    // Accumulate partial histograms
    for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE) 
    {
        sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM64_BIN_COUNT];
    }

    data[threadIdx.x] = sum;
    __syncthreads(); // Synchronize threads before reduction

    // Optimize reduction using loop unrolling
    for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1) 
    {
        if (threadIdx.x < stride) 
        {
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
        __syncthreads(); // Sync threads at the end of each stride
    }

    if (threadIdx.x == 0) 
    {
        d_Histogram[blockIdx.x] = data[0]; // Output result to global memory
    }
}