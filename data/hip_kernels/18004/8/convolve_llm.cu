#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolve(unsigned char* imgInput, float* mask, int width, int height, int paddingX, int paddingY, int kWidth, int kHeight, unsigned char* imgOutput)
{
    // Calculate our pixel's location
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    // Load kernel size
    float sum = 0.0;
    int pWidth = kWidth / 2;
    int pHeight = kHeight / 2;

    // Execute only for valid pixels
    if (x >= pWidth + paddingX && y >= pHeight + paddingY && x < (blockDim.x * gridDim.x) - pWidth - paddingX &&
        y < (blockDim.y * gridDim.y) - pHeight - paddingY)
    {
#pragma unroll // Unroll the loop for performance
        for (int j = -pHeight; j <= pHeight; j++)
        {
            for (int i = -pWidth; i <= pWidth; i++)
            {
                // Sample the weight for this location
                int ki = i + pWidth;
                int kj = j + pHeight;
                float w = mask[kj * kWidth + ki];

                // Accumulate the result
                sum += w * float(imgInput[(y + j) * width + (x + i)]);
            }
        }
    }
    
    // Store the computed result
    imgOutput[y * width + x] = static_cast<unsigned char>(sum);
}