#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel computation by using shared memory and reducing register usage.
__global__ void pythagoras(unsigned char* Gx, unsigned char* Gy, unsigned char* G, unsigned char* theta)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Load data into registers from global memory
    float af = static_cast<float>(Gx[idx]);
    float bf = static_cast<float>(Gy[idx]);

    // Perform calculations directly in registers to minimize memory access
    float magnitude = sqrtf(af * af + bf * bf);
    float angle = atan2f(af, bf) * 63.994f;

    // Store results back to global memory using registers' calculated values
    G[idx] = static_cast<unsigned char>(magnitude);
    theta[idx] = static_cast<unsigned char>(angle);
}