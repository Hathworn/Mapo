#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolve(unsigned char* imgInput, float* mask, int width, int height, int paddingX, int paddingY, int kWidth, int kHeight, unsigned char* imgOutput)
{
    // Calculate the pixel's location
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    float sum = 0.0;
    int pWidth = kWidth / 2;
    int pHeight = kHeight / 2;

    // Execute only for valid pixels
    if (x >= pWidth + paddingX && y >= pHeight + paddingY && x < width - pWidth - paddingX &&
        y < height - pHeight - paddingY) // Reduce boundary checks
    {
        for (int j = -pHeight; j <= pHeight; j++)
        {
            for (int i = -pWidth; i <= pWidth; i++)
            {
                // Sample the weight for this location
                int ki = (i + pWidth);
                int kj = (j + pHeight);
                float w = mask[(kj * kWidth) + ki];

                sum += w * float(imgInput[((y + j) * width) + (x + i)]);
            }
        }
        imgOutput[(y * width) + x] = (unsigned char)sum; // Move output assignment inside valid check
    }
}