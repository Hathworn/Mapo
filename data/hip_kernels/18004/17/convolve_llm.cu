#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolve(unsigned char* imgInput, int width, int height, int paddingX, int paddingY, int kWidth, int kHeight, unsigned int offset, unsigned char* imgOutput)
{
    // Calculate our pixel's location
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x >= paddingX && y >= paddingY && x < width - paddingX && y < height - paddingY) // Validate pixel is within bounds
    {
        float sum = 0.0;
        int pWidth = kWidth / 2;
        int pHeight = kHeight / 2;

        // Iterate over the kernel
        for (int j = -pHeight; j <= pHeight; j++)
        {
            for (int i = -pWidth; i <= pWidth; i++)
            {
                // Calculate mask index and ensure bounds are valid
                int ki = i + pWidth;
                int kj = j + pHeight;
                float w = mask[offset + kj * kWidth + ki];

                // Accumulate weighted pixel value
                sum += w * float(imgInput[(y + j) * width + (x + i)]);
            }
        }
        imgOutput[y * width + x] = (unsigned char)sum;
    }
}