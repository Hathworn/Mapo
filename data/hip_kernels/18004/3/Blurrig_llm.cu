#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLUR_SIZE 1

__global__ void Blurrig(unsigned char* output, unsigned char* input, int height, int width) {
    // Calculate thread position within the grid
    int Col = threadIdx.x + blockIdx.x * blockDim.x;
    int Row = threadIdx.y + blockIdx.y * blockDim.y;

    if (Col < width && Row < height) {
        int pixVal = 0;
        int pixels = 0;

        // Optimize loop calculations by pre-calculating bounds
        int rowStart = max(0, Row - BLUR_SIZE);
        int rowEnd = min(height - 1, Row + BLUR_SIZE);
        int colStart = max(0, Col - BLUR_SIZE);
        int colEnd = min(width - 1, Col + BLUR_SIZE);

        for (int curRow = rowStart; curRow <= rowEnd; ++curRow) {
            for (int curCol = colStart; curCol <= colEnd; ++curCol) {
                pixVal += input[curRow * width + curCol];
                pixels++; // count valid pixels
            }
        }
        // Write blurred pixel value
        output[Row * width + Col] = (unsigned char)(pixVal / pixels);
    }
}