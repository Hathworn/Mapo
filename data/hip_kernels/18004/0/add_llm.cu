#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Improved order for better readability
    if (index < SIZE) {
        c[index] = a[index] + b[index]; // Maintain straightforward element addition
    }
}