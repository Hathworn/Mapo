#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void greyConvertor(unsigned char* output, uchar3 const* input, const uint height, const uint width) {
    // Pre-calculate the pixel index using linear thread indexing for better performance
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if thread is within the image bounds
    if (x < width && y < height) {
        int grayOffset = y * width + x;
        uchar3 pixel = input[grayOffset];  // Fetch the pixel data once
        // Apply weighted sum for grayscale conversion with float multiplication for precision
        output[grayOffset] = 0.21f * pixel.x + 0.72f * pixel.y + 0.07f * pixel.z;
    }
}