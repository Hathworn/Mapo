#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Convolution_2D_globalMemory(unsigned char* imgInput, unsigned char* imgOutput, const float* mask, int height, int width, int channels) {

    // Use shared memory to reduce global memory accesses
    __shared__ float sharedMask[MASK_WIDTH * MASK_WIDTH];

    int tx = threadIdx.x, ty = threadIdx.y;
    int Row = blockIdx.x * blockDim.x + tx;
    int Col = blockIdx.y * blockDim.y + ty;

    if (tx < MASK_WIDTH && ty < MASK_WIDTH) {
        sharedMask[tx * MASK_WIDTH + ty] = mask[tx * MASK_WIDTH + ty];
    }
    __syncthreads(); // Synchronize to ensure mask is loaded

    if (Row < height && Col < width) {
        for (int c = 0; c < channels; c++) {
            float sum = 0.0f;
            for (int i = 0; i < MASK_WIDTH; i++) {
                int filterRow = Row - MASK_WIDTH / 2 + i;
                for (int j = 0; j < MASK_WIDTH; j++) {
                    int filterCol = Col - MASK_WIDTH / 2 + j;

                    if ((filterRow >= 0) && (filterRow < height) && (filterCol >= 0) && (filterCol < width)) {
                        sum += imgInput[(filterRow * width + filterCol) * channels + c] * sharedMask[i * MASK_WIDTH + j];
                    }
                }
            }
            sum /= MASK_WIDTH * MASK_WIDTH;
            imgOutput[(Row * width + Col) * channels + c] = (unsigned char)sum;
        }
    }
}