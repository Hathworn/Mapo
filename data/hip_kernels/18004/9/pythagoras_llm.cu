#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pythagoras(unsigned char* Gx, unsigned char* Gy, unsigned char* G, unsigned char* theta)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global index

    // Use local variables for improved register use and readability
    float af = static_cast<float>(Gx[idx]);
    float bf = static_cast<float>(Gy[idx]);

    // Calculate gradient magnitude and orientation
    float magnitude = sqrtf(af * af + bf * bf);
    float angle = atan2f(af, bf) * 63.994f;

    // Store results
    G[idx] = static_cast<unsigned char>(magnitude); 
    theta[idx] = static_cast<unsigned char>(angle);
}