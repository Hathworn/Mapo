#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pythagoras(unsigned char* Gx, unsigned char* Gy, unsigned char* G, unsigned char* theta)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Use shared memory if needed (size calculation would depend on further context)
    // __shared__ float sharedGx[BlockSize];
    // __shared__ float sharedGy[BlockSize];

    // Load data from global to registers
    float af = float(Gx[idx]);
    float bf = float(Gy[idx]);

    // Compute sqrt and atan2 using registers for intermediate steps
    float magnitude = sqrtf(af * af + bf * bf);
    float angle = atan2f(af, bf) * 63.994f;

    // Store results back to global memory
    G[idx] = (unsigned char)magnitude;
    theta[idx] = (unsigned char)angle;
}