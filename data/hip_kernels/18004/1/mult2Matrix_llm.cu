#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult2Matrix(float *M, float *N, float *P) {
    // Shared memory for tiles of matrices M and N
    __shared__ float tileM[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float tileN[BLOCK_SIZE][BLOCK_SIZE];

    // Calculate the row and column indices of P
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    
    float Pvalue = 0;

    // Loop over the tiles of the input matrices
    for (int t = 0; t < (WIDTH + BLOCK_SIZE - 1) / BLOCK_SIZE; ++t) {
        // Collaborative loading of tiles into shared memory
        if (Row < WIDTH && t * BLOCK_SIZE + threadIdx.x < WIDTH)
            tileM[threadIdx.y][threadIdx.x] = M[Row * WIDTH + t * BLOCK_SIZE + threadIdx.x];
        else
            tileM[threadIdx.y][threadIdx.x] = 0.0;

        if (Col < WIDTH && t * BLOCK_SIZE + threadIdx.y < WIDTH)
            tileN[threadIdx.y][threadIdx.x] = N[(t * BLOCK_SIZE + threadIdx.y) * WIDTH + Col];
        else
            tileN[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        // Multiply the tiles together
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Pvalue += tileM[threadIdx.y][k] * tileN[k][threadIdx.x];
        }
        __syncthreads();
    }

    // Write result to matrix P
    if (Row < WIDTH && Col < WIDTH) {
        P[Row * WIDTH + Col] = Pvalue;
    }
}