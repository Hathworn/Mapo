#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grayScale(unsigned char* imgInput, unsigned char* imgOutput, int Row, int Col) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < Row && col < Col) {  // Correct boundary check
        int idx = (row * Col + col) * 3;  // Precompute index
        imgOutput[row * Col + col] = imgInput[idx + 2] * 0.299f + imgInput[idx + 1] * 0.587f + imgInput[idx] * 0.114f;  // Use float constants
    }
}