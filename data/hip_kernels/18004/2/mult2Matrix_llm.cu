#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult2Matrix(float *M, float *N, float *P) {
    __shared__ float shared_m_tile[TILE_WIDTH][TILE_WIDTH]; // Use float for matching input matrix type
    __shared__ float shared_n_tile[TILE_WIDTH][TILE_WIDTH]; // Use float for matching input matrix type

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int col = blockIdx.x * blockDim.x + tx;
    int row = blockIdx.y * blockDim.y + ty;
    
    if (row < WIDTH && col < WIDTH) {
        float result = 0;
        
        for (int phase = 0; phase < (WIDTH + TILE_WIDTH - 1) / TILE_WIDTH; phase++) { // Ensure loop handles full range

            // Load tiles from global memory to shared memory, handle edge cases
            if (row < WIDTH && (phase * TILE_WIDTH + tx) < WIDTH)
                shared_m_tile[ty][tx] = M[row * WIDTH + phase * TILE_WIDTH + tx];
            else
                shared_m_tile[ty][tx] = 0.0;
                
            if (col < WIDTH && (phase * TILE_WIDTH + ty) < WIDTH)
                shared_n_tile[ty][tx] = N[(phase * TILE_WIDTH + ty) * WIDTH + col];
            else
                shared_n_tile[ty][tx] = 0.0;

            __syncthreads();

            for (int k = 0; k < TILE_WIDTH; k++) {
                result += shared_m_tile[ty][k] * shared_n_tile[k][tx];
            }
            __syncthreads();
        }
        P[row * WIDTH + col] = result;
    }
}