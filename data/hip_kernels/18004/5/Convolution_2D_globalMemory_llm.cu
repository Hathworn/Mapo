#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define MASK_WIDTH 3

__global__ void Convolution_2D_globalMemory(unsigned char* imgInput, unsigned char* imgOutput, const float* mask, int height, int width, int channels) {
    // Calculate the row and column indexes for the current pixel
    int rows = blockIdx.x * blockDim.x + threadIdx.x;
    int cols = blockIdx.y * blockDim.y + threadIdx.y;

    // Check boundaries
    if (rows < height && cols < width) {
        for (int c = 0; c < channels; c++) {
            float sum = 0;
            // Perform convolution operation
            for (int i = 0; i < MASK_WIDTH; i++) {
                int filterRow = rows - MASK_WIDTH / 2 + i;
                for (int j = 0; j < MASK_WIDTH; j++) {
                    int filterCol = cols - MASK_WIDTH / 2 + j;
                    if (filterRow >= 0 && filterRow < height && filterCol >= 0 && filterCol < width) {
                        sum += imgInput[(filterRow * width + filterCol) * channels + c] * mask[i * MASK_WIDTH + j];
                    }
                }
            }
            // Assign the result to the output image
            imgOutput[(rows * width + cols) * channels + c] = (unsigned char)sum;
        }
    }
}