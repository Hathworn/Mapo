#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void staticReverse(int *d, int n)
{
    // Allocate shared memory dynamically to handle arrays larger than 64 elements
    extern __shared__ int s[];
    int t = threadIdx.x;
    int tr = n-t-1;
    
    // Ensure threads do not access out-of-bound memory
    if (t < n) 
    {
        s[t] = d[t];
    }
    __syncthreads();
    
    if (t < n) 
    {
        d[t] = s[tr];
    }
}