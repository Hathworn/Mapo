#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dynamicReverse(int *d, int n)
{
    extern __shared__ int s[];
    int t = threadIdx.x;
    int tr = n - t - 1;

    // Load input into shared memory, ensuring we don't out-of-bounds access
    if (t < n) {
        s[t] = d[t];
    }
    __syncthreads();

    // Store reversed data from shared memory back into global memory
    if (t < n) {
        d[t] = s[tr];
    }
}