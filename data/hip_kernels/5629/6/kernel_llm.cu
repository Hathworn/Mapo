#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *a, int offset)
{
    int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
    float x = static_cast<float>(i);

    // Precompute sinf(x) and cosf(x) for efficiency
    float s = sincosf(x, &c);
    
    // Utilize fmaf for more precise and efficient computation
    a[i] = fmaf(s, s, c * c);
}