#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double dnorm(float x, float mu, float sigma)
{
    float std = (x - mu)/sigma;
    float e = exp(-0.5 * std * std);
    return e / (sigma * sqrt(2 * 3.141592653589793));
}

__global__ void log_truncNorm(float *out, float *unifVals, int N)
{
    // Combine block and thread indices to form global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        // Calculate log of value scaled by density
        out[idx] = log(unifVals[idx]) * dnorm(unifVals[idx], 0, 1) / 0.3413447460685;
    }
}