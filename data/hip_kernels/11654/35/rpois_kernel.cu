#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rpois_kernel(hiprandState *state, int *vals, int n, double lambda)
{
// Usual block/thread indexing...
int myblock = blockIdx.x + blockIdx.y * gridDim.x;
int blocksize = blockDim.x * blockDim.y * blockDim.z;
int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
int idx = myblock * blocksize + subthread;

if (idx < n) {
vals[idx] = hiprand_poisson(&state[idx], lambda);
}
return;
}