#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rpois_kernel(hiprandState *state, int *vals, int n, double lambda)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Process valid index
    if (idx < n) {
        vals[idx] = hiprand_poisson(&state[idx], lambda);
    }
}