#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_kernel(double *vals, double *ans, int N, double mu)
{
    // Calculate global thread index more concisely
    int idx = blockIdx.x * blockDim.x * blockDim.y * blockDim.z 
            + blockIdx.y * gridDim.x * blockDim.x * blockDim.y * blockDim.z 
            + threadIdx.z * blockDim.y * blockDim.x 
            + threadIdx.y * blockDim.x 
            + threadIdx.x;

    if(idx < N)
        ans[idx] = vals[idx] + mu;
}