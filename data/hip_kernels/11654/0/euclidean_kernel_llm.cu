#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void euclidean_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // If an element is to be computed
    if (x < n_a && y < n_b) {

        extern __shared__ float temp[];  // Use dynamic shared memory allocation

        float sum = 0.0;  // Use a register to accumulate partial sums

        for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            float t = vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset];
            sum += (t * t);
        }

        // Thread's partial result is stored into shared memory
        temp[threadIdx.x] = sum;

        __syncthreads();

        // Perform parallel reduction using thread registers
        for (size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Write the result to global memory
        if (threadIdx.x == 0) {
            d[y * pitch_d + x] = sqrt(temp[0]);
        }
    }
}