#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fadd_kernel(float *vals, int N, float mu)
{
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x 
            + blockIdx.y * (gridDim.x * blockDim.x) * blockDim.y 
            + threadIdx.y * gridDim.x * blockDim.x;

    // Perform operation only within bounds
    if(idx < N)
        vals[idx] = mu;
}