#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double dnorm(float x, float mu, float sigma)
{
    float std = (x - mu) / sigma;
    float e = exp(-0.5 * std * std);
    return (e / (sigma * sqrt(2 * 3.141592653589793)));
}

__global__ void dnorm_kernel(float *vals, int N, float mu, float sigma)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if idx is within bounds
    if (idx < N) {
        // Calculate the value using dnorm and store it in vals
        vals[idx] = dnorm(vals[idx], mu, sigma);
    }
}