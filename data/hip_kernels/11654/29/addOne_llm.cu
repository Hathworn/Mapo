#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void addOne(float *vals, int N, float *out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplified calculation of global index

    if (idx < N) {
        out[idx] = vals[idx] + 1.0f; // Use float suffix for precision
    }
}