#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void euclidean_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x, y = blockIdx.y;
    if ((x == y) && (x < n_a) && (threadIdx.x == 0))
        d[y * pitch_d + x] = 0.0;

    // Optimize by using fewer __syncthreads thanks to warp shuffle
    if (y < n_a && x < y) {
        float sum = 0.0;
        for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            float t = vg_a[x * pitch_a + offset] - vg_a[y * pitch_a + offset];
            sum += (t * t);
        }

        // Reduce sum using warp reduction
        for (int stride = warpSize / 2; stride > 0; stride /= 2) {
            sum += __shfl_down_sync(0xFFFFFFFF, sum, stride);
        }

        // Use block-level reduction for final result
        __shared__ float warp_sums[32];
        if (threadIdx.x % warpSize == 0) {
            warp_sums[threadIdx.x / warpSize] = sum;
        }
        __syncthreads();

        if (threadIdx.x < blockDim.x / warpSize) {
            sum = warp_sums[threadIdx.x];
            for (int stride = (blockDim.x / warpSize) / 2; stride > 0; stride /= 2) {
                sum += __shfl_down_sync(0xFFFFFFFF, sum, stride);
            }
        }

        if (threadIdx.x == 0) {
            float s = sqrt(warp_sums[0]);
            d[y * pitch_d + x] = s;
            d[x * pitch_d + y] = s;
        }
    }
}