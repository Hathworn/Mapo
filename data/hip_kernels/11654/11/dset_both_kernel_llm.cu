#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dset_both_kernel(double *vals, int N, double mu, float sd)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x * blockDim.y * blockDim.z + 
              blockIdx.y * blockDim.x * blockDim.z + 
              blockIdx.z * blockDim.x + 
              threadIdx.x;
              
    // Check bounds and set value
    if (idx < N)
        vals[idx] = mu + sd;
}