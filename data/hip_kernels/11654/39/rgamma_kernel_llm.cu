#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rgamma_kernel(hiprandState *state, int state_len, float *vals, int n, float a, float scale)
{
    // Precomputed constants
    const float sqrt32 = 5.656854;
    const float exp_m1 = 0.36787944117144232159;
    const float q_coef[] = {0.04166669, 0.02083148, 0.00801191, 0.00144121, -7.388e-5, 2.4511e-4, 2.424e-4};
    const float a_coef[] = {0.3333333, -0.250003, 0.2000062, -0.1662921, 0.1423657, -0.1367177, 0.1233795};

    // Computing thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 
              (blockIdx.y * gridDim.x) * blockDim.x * gridDim.x + 
              threadIdx.y * blockDim.x * blockDim.y + threadIdx.z * blockDim.x * blockDim.y * blockDim.z;

    if (idx >= state_len) return;

    // Calculate the number of extra and regular threads
    int n_extra = n % state_len;
    int n_lo = n / state_len;
    int n_hi = n_lo + 1;
    int out_idx_start = (idx < n_extra) ? n_hi * idx : n_hi * n_extra + n_lo * (idx - n_extra);
    int n_per_thread = (idx < n_extra) ? n_hi : n_lo;

    float aa = 0.0, aaa = 0.0, s, s2, d, q0, b, si, c;
    float e, p, q, r, t, u, v, x, ret_val;

    // Generate RNG's for each thread
    for (int gen_num = 0; gen_num < n_per_thread; gen_num++)
    {
        if (a < 1.0) { // GS algorithm for a < 1
            if (a == 0) {
                vals[out_idx_start + gen_num] = 0.0;
                continue;
            }
            e = 1.0 + exp_m1 * a;
            do {
                p = e * hiprand_normal(&state[idx]);
                if (p >= 1.0) {
                    x = -logf((e - p) / a);
                    if (logf(hiprand_uniform(&state[idx])) <= (1.0 - a) * logf(x)) {
                        break;
                    }
                } else {
                    x = expf(logf(p) / a);
                    if (logf(hiprand_uniform(&state[idx])) <= x) {
                        break;
                    }
                }
            } while (true);
            vals[out_idx_start + gen_num] = scale * x;
            continue;
        }

        if (a != aa) {
            aa = a;
            s2 = a - 0.5;
            s = sqrtf(s2);
            d = sqrt32 - s * 12.0;
        }

        t = hiprand_normal(&state[idx]);
        x = s + 0.5 * t;
        ret_val = x * x;

        if (t >= 0.0) {
            vals[out_idx_start + gen_num] = scale * ret_val;
            continue;
        }

        u = hiprand_uniform(&state[idx]);
        if ((d * u) <= (t * t * t)) {
            vals[out_idx_start + gen_num] = scale * ret_val;
            continue;
        }

        if (a != aaa) {
            aaa = a;
            r = 1.0 / a;
            q0 = ((((((q_coef[6] * r + q_coef[5]) * r + q_coef[4]) * r + q_coef[3]) * r + q_coef[2]) * r + q_coef[1]) * r + q_coef[0]) * r;
            
            if (a <= 3.686) {
                b = 0.463 + s + 0.178 * s2;
                si = 1.235;
                c = 0.195 / s - 0.079 + 0.16 * s;
            } else if (a <= 13.022) {
                b = 1.654 + 0.0076 * s2;
                si = 1.68 / s + 0.275;
                c = 0.062 / s + 0.024;
            } else {
                b = 1.77;
                si = 0.75;
                c = 0.1515 / s;
            }
        }

        if (x > 0.0) {
            v = t / (s + s);
            q = (fabs(v) <= 0.25) ? q0 + 0.5 * t * t * ((((a_coef[6] * v + a_coef[5]) * v + a_coef[4]) * v + a_coef[3]) * v + a_coef[2]) * v + a_coef[1] * v + a_coef[0] * v : q0 - s * t + 0.25 * t * t + (s2 + s2) * log(1.0 + v);

            if (logf(1.0 - u) <= q) {
                vals[out_idx_start + gen_num] = scale * ret_val;
                continue;
            }
        }

        while (true) {
            e = -logf(hiprand_uniform(&state[idx]));
            u = hiprand_uniform(&state[idx]) * 2.0 - 1.0;
            t = (u < 0.0) ? b - si * e : b + si * e;

            if (t >= -0.71874483771719) {
                v = t / (s + s);
                q = (fabs(v) <= 0.25) ? q0 + 0.5 * t * t * ((((a_coef[6] * v + a_coef[5]) * v + a_coef[4]) * v + a_coef[3]) * v + a_coef[2]) * v + a_coef[1] * v + a_coef[0] * v : q0 - s * t + 0.25 * t * t + (s2 + s2) * log(1.0 + v);

                if (q > 0.0) {
                    float w = expm1f(q);
                    if (c * fabs(u) <= w * expf(e - 0.5 * t * t)) {
                        break;
                    }
                }
            }
        }
        x = s + 0.5 * t;
        vals[out_idx_start + gen_num] = scale * x * x;
    }
}