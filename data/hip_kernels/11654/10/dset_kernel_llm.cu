#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dset_kernel(double *vals, int N, double mu)
{
    // Calculate unique thread index more efficiently
    int idx = blockIdx.x * blockDim.x + threadIdx.x +
              blockIdx.y * blockDim.y * gridDim.x +
              blockIdx.z * blockDim.z * gridDim.x * gridDim.y;

    if (idx < N) {
        vals[idx] = mu; // Assign value if within bounds
    }
}