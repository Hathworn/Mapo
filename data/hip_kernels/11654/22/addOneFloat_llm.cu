#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addOneFloat(double* vals, int N, float *out)
{
    // Flattened thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x 
            + blockIdx.y * blockDim.x * gridDim.x * blockDim.y + threadIdx.y * blockDim.x * gridDim.x 
            + threadIdx.z * blockDim.x * blockDim.y * gridDim.x;

    if(idx < N) {
        // Type cast double to float and add 1.0
        out[idx] = static_cast<float>(vals[idx]) + 1.0f;
    }
}