#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void canberra_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // Use grid-stride loop for better performance
    for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < n_a * n_b; idx += gridDim.x * blockDim.x) {
        size_t x = idx / n_b;
        size_t y = idx % n_b;

        if (x < n_a && y < n_b) {
            __shared__ float temp[NUM_THREADS];

            // Use a single variable for accumulation
            float localSum = 0.0f;
            for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
                float num = abs(vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset]);
                float den = abs(vg_a[x * pitch_a + offset] + vg_b[y * pitch_b + offset]);
                if (den != 0.0f) {
                    localSum += num / den;
                }
            }
            temp[threadIdx.x] = localSum;

            __syncthreads();

            // Use warp reduction for better efficiency
            for (size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
                if (threadIdx.x < stride) {
                    temp[threadIdx.x] += temp[threadIdx.x + stride];
                }
                __syncthreads();
            }

            if (threadIdx.x == 0) {
                d[y * pitch_d + x] = temp[0];
            }
        }
    }
}