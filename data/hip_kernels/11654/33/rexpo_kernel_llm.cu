#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rexpo_kernel(hiprandState *state, float *vals, int n, float lambda) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplified 1D indexing

    // Ensure idx is within bounds and perform calculation
    if (idx < n) {
        vals[idx] = -log(hiprand_uniform(&state[idx])) / lambda;
    }
}