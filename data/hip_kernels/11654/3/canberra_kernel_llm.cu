#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void canberra_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // If all elements need to be computed
    if(x < n_a && y < n_b) {
        extern __shared__ float temp[];

        temp[threadIdx.x] = 0.0f; // Use 0.0f for float
        float sum = 0.0f; // Optimized local accumulation

        for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            float a_val = vg_a[x * pitch_a + offset];
            float b_val = vg_b[y * pitch_b + offset];
            float num = fabsf(a_val - b_val);
            float den = fabsf(a_val + b_val);
            if(den != 0.0f) {
                sum += num / den; // Accumulate locally
            }
        }

        temp[threadIdx.x] = sum; // Store local sum
        __syncthreads();

        // Use warp-level primitives for reduction
        for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if(threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Write to global memory
        if(threadIdx.x == 0) {
            d[y * pitch_d + x] = temp[0];
        }
    }
}