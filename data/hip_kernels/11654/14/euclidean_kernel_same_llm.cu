#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void euclidean_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x, y = blockIdx.y;

    if ((x == y) && (x < n_a) && (threadIdx.x == 0))
        d[y * pitch_d + x] = 0.0;

    // If all element computation is needed
    if (y < n_a && x < y) {
        __shared__ float temp[NUM_THREADS];

        // Initialize shared memory
        temp[threadIdx.x] = 0.0f;

        // Compute squared differences
        for (size_t offset = threadIdx.x; offset < k; offset += NUM_THREADS) {
            float t = vg_a[x * pitch_a + offset] - vg_a[y * pitch_a + offset];
            temp[threadIdx.x] += t * t;
        }

        // Synchronize threads
        __syncthreads();

        // Reduction within a block
        for (size_t stride = blockDim.x / 2; stride > 0; stride /= 2) {
            if (threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Write the result to global memory
        if (threadIdx.x == 0) {
            float s = sqrtf(temp[0]);  // Use sqrtf for single precision
            d[y * pitch_d + x] = s;
            d[x * pitch_d + y] = s;
        }
    }
}