#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double dnorm(float x, float mu, float sigma)
{
    float std = (x - mu)/sigma;
    float e = exp( - 0.5 * std * std);
    return (e / ( sigma * sqrt(2 * 3.141592653589793)));
}

__global__ void dnorm_kernel(float *vals, int N, float mu, float sigma)
{
    // Calculate global thread index directly for improved readability.
    int idx = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.x * gridDim.x) + (threadIdx.y * blockDim.x * blockDim.y) + (threadIdx.z * blockDim.x * blockDim.y * blockDim.z);

    if (idx < N) {
        float std = (vals[idx] - mu)/sigma;
        float e = exp( - 0.5 * std * std);
        vals[idx] = e / ( sigma * sqrt(2 * 3.141592653589793));
    }
}