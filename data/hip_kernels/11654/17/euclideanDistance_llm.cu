#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void euclideanDistance(const float *data_a, int nrow_a, const float *data_b, int nrow_b, int ncol, float *ans)
{
    // Optimize thread indexing for coalesced memory access
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i < nrow_a && j < nrow_b) {
        int off_a = i;
        int off_b = j;
        float sum = 0;
        
        // Loop unrolling for optimization
        int k = 0;
        for(; k <= ncol - 4; k += 4, off_a += 4 * nrow_a, off_b += 4 * nrow_b) {
            float tmp1 = data_a[off_a] - data_b[off_b];
            float tmp2 = data_a[off_a + nrow_a] - data_b[off_b + nrow_b];
            float tmp3 = data_a[off_a + 2 * nrow_a] - data_b[off_b + 2 * nrow_b];
            float tmp4 = data_a[off_a + 3 * nrow_a] - data_b[off_b + 3 * nrow_b];
            sum += tmp1 * tmp1 + tmp2 * tmp2 + tmp3 * tmp3 + tmp4 * tmp4;
        }
        
        // Handling the remaining elements
        for(; k < ncol; k++, off_a += nrow_a, off_b += nrow_b) {
            float tmp = data_a[off_a] - data_b[off_b];
            sum += tmp * tmp;
        }
        ans[i + j * nrow_a] = sqrt(sum);
    }
}