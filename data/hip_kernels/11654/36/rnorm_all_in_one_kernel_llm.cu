#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rnorm_all_in_one_kernel(float *vals, int n, float mu, float sigma)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 
              blockIdx.y * (blockDim.x * gridDim.x) + 
              threadIdx.y * gridDim.x * blockDim.x + 
              blockIdx.z * (blockDim.x * blockDim.y * gridDim.x * gridDim.y) + 
              threadIdx.z * (gridDim.x * gridDim.y * blockDim.x * blockDim.y);

    // Setup the RNG only for valid indices
    if (idx < n) {
        hiprandState rng_state;
        hiprand_init(9131 + idx * 17, 0, 0, &rng_state);
        vals[idx] = mu + sigma * hiprand_normal(&rng_state);
    }
}