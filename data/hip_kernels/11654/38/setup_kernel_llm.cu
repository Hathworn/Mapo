#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_kernel(hiprandState *state, int *seeds, int n)
{
    // Directly use built-in variables to calculate global thread index
    int idx = blockIdx.x * blockDim.x * blockDim.y * blockDim.z + 
              blockIdx.y * blockDim.x * blockDim.y * blockDim.z + 
              threadIdx.z * blockDim.x * blockDim.y + 
              threadIdx.y * blockDim.x + 
              threadIdx.x;
              
    // Check and initialize state using atomic operations
    if (idx < n)
    {
        hiprand_init(seeds[idx], idx, 0, &state[idx]);
    }
}