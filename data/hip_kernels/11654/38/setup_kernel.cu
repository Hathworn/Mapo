#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_kernel(hiprandState  *state, int *seeds, int n)
{
// Usual block/thread indexing...
int myblock = blockIdx.x + blockIdx.y * gridDim.x;
int blocksize = blockDim.x * blockDim.y * blockDim.z;
int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
int idx = myblock * blocksize + subthread;
if (idx < n){
hiprand_init(seeds[idx], idx, 0, &state[idx]);
}
return;
}