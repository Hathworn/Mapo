#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void euclidean_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // If an element is to be computed
    if(x < n_a && y < n_b) {

        // Use dynamically allocated shared memory
        extern __shared__ float temp[];

        temp[threadIdx.x] = 0.0f;

        // Unrolling the loop for performance optimization
        size_t offset;
        for(offset = threadIdx.x; offset + 4 < k; offset += blockDim.x * 4) {
            float t0 = vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset];
            float t1 = vg_a[x * pitch_a + offset + 1] - vg_b[y * pitch_b + offset + 1];
            float t2 = vg_a[x * pitch_a + offset + 2] - vg_b[y * pitch_b + offset + 2];
            float t3 = vg_a[x * pitch_a + offset + 3] - vg_b[y * pitch_b + offset + 3];
            temp[threadIdx.x] += t0 * t0 + t1 * t1 + t2 * t2 + t3 * t3;
        }

        // Process any remaining elements
        for(; offset < k; ++offset) {
            float t = vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset];
            temp[threadIdx.x] += t * t;
        }
        
        // Sync with other threads
        __syncthreads();

        // Reduce using shared memory
        for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if(threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Write to global memory
        if(threadIdx.x == 0) {
            d[y * pitch_d + x] = sqrtf(temp[0]);
        }
    }
}