#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setValue_kernel(int *vals, int N)
{
    // Calculate linear index using 3D grid and 3D blocks
    int idx = blockIdx.x * blockDim.x * blockDim.y * blockDim.z + 
              threadIdx.z * blockDim.x * blockDim.y + 
              threadIdx.y * blockDim.x + 
              threadIdx.x;

    // Check if within bounds and set value
    if (idx < N)
    {
        vals[idx] = idx;
    }
}