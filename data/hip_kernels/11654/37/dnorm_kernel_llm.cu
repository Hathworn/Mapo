#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double dnorm(float x, float mu, float sigma)
{
    float std = (x - mu) / sigma;
    float e = exp(-0.5f * std * std);
    return e / (sigma * sqrtf(2.0f * 3.141592653589793f));
}

__global__ void dnorm_kernel(float *vals, int N, float mu, float sigma)
{
    // Calculate global index directly
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        // Inline the calculation, reducing function call overhead
        float std = (vals[idx] - mu) / sigma;
        float e = exp(-0.5f * std * std);
        vals[idx] = e / (sigma * sqrtf(2.0f * 3.141592653589793f));
    }
}