#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double dnorm(float x, float mu, float sigma) {
    float std = (x - mu) / sigma;
    float e = exp(-0.5f * std * std);
    return e / (sigma * sqrtf(2.0f * 3.141592653589793f));
}

__global__ void truncNorm(float *out, float *unifVals, int N) {
    // Combine block and thread indices into a single linear index
    int idx = blockIdx.x * blockDim.x * blockDim.y * blockDim.z + 
              threadIdx.z * blockDim.x * blockDim.y + 
              threadIdx.y * blockDim.x + 
              threadIdx.x;
    
    if (idx < N) {
        // Utilize precomputed constant instead of inline calculation
        const float inv_norm_const = 1.0f / 0.3413447460685f;
        out[idx] = unifVals[idx] * dnorm(unifVals[idx], 0, 1) * inv_norm_const;
    }
}