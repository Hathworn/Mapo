#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use a 1D block and grid for simplified indexing
__global__ void runif_kernel(hiprandState *state, float *vals, int n, float lo, float hi)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Compute global index
    if (idx < n) {
        vals[idx] = lo + (hi - lo) * hiprand_uniform(&state[idx]); // Generate random number
    }
}