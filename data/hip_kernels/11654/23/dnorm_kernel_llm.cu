#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double dnorm(float x, float mu, float sigma)
{
    float std = (x - mu)/sigma;
    float e = exp( - 0.5 * std * std);
    return(e / ( sigma * sqrt(2 * 3.141592653589793)));
}

__global__ void dnorm_kernel(float *vals, int N, float mu, float sigma, float *out)
{
    // Combine calculations to avoid redundant computation
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float std = (vals[idx] - mu) / sigma;
        float e = exp( -0.5f * std * std);
        out[idx] = e / (sigma * 2.50662827463f);  // precomputed sqrt(2 * pi)
    }
}