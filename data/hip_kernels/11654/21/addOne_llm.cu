#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addOne(double* vals, int N)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x 
            + blockIdx.y * gridDim.x * blockDim.x * blockDim.y 
            + threadIdx.y * blockDim.x;
    
    // Ensure thread is within bounds
    if(idx < N) {
        vals[idx] += 1.0; // Increment value by 1.0
    }
}