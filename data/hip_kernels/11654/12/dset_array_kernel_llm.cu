#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dset_array_kernel(double *vals, int N, double *mu)
{
    // Calculate global index using the built-in function
    int idx = blockIdx.x * blockDim.x + threadIdx.x
            + blockIdx.y * gridDim.x * blockDim.x * blockDim.y
            + blockIdx.z * gridDim.x * gridDim.y * blockDim.x * blockDim.y * threadIdx.z;

    // Perform the operation only if within bounds
    if(idx < N)
        vals[idx] = mu[0];
}