#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double dnorm(float x, float mu, float sigma)
{
    float std = (x - mu) / sigma;
    float e = exp(-0.5 * std * std);
    return (e / (sigma * sqrt(2 * 3.141592653589793)));
}

__global__ void dnorm_kernel(float *vals, int N, float mu, float sigma)
{
    // Optimize thread index computation
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (idx < N) {
        float std = (vals[idx] - mu) / sigma;
        float e = exp(-0.5 * std * std);
        vals[idx] = log(e / (sigma * sqrt(2 * 3.141592653589793)));
    }
}