#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel for random normal numbers generation
__global__ void rnorm_kernel(hiprandState *state, float *vals, int n, float mu, float sigma)
{
    // 1D block/thread indexing
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process within bounds
    if (idx < n) {
        // Generate a normal random number and store it
        vals[idx] = mu + sigma * hiprand_normal(&state[idx]);
    }
}