#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void minkowski_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x, y = blockIdx.y;

    // Check if the current thread should perform computations
    if (x < n_a && y < n_b) {
        extern __shared__ float temp[];

        // Initialize shared memory
        temp[threadIdx.x] = 0.0;
        
        // Compute partial sum of powers
        for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            float t = fabsf(vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset]);
            temp[threadIdx.x] += __powf(t, p);
        }

        // Sync with other threads
        __syncthreads();

        // Improved reduction using warp, avoiding unnecessary sync
        for (size_t stride = warpSize; stride >= 1; stride /= 2) {
            if (threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            }
            __syncwarp();
        }

        // Write to global memory
        if (threadIdx.x == 0) {
            float power = 1.f / p;
            d[y * pitch_d + x] = __powf(temp[0], power);
        }
    }
}