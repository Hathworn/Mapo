#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _reluforw(int n, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Efficient loop increment using grid-stride loop
    for (; i < n; i += blockDim.x * gridDim.x) {
        if (y[i] < 0) y[i] = 0;
    }
}