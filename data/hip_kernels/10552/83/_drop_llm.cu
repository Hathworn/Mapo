#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _drop(int n, float *x, float *xmask, float dropout, float scale) {
    // Use a stride loop and avoid recalculating i
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (; i < n; i += stride) {
        if (xmask[i] < dropout) {
            x[i] = 0;
        } else {
            x[i] *= scale;
        }
    }
}