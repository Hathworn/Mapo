#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized _kgauss32map kernel
__global__ void _kgauss32map(int nx, int ns, float *x2, float *s2, float *k, float g) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int n = nx * ns;

    // Loop through elements with stride
    for (; i < n; i += stride) {
        int xi = i % nx;  // Compute row index
        int si = i / nx;  // Compute column index
        // Update k[i] using precomputed indices and exponential function
        k[i] = expf(-g * (x2[xi] + s2[si] - 2 * k[i]));
    }
}