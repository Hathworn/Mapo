#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _kgauss32(int nx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k, float g) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int n = nx * ns;

    // Use shared memory for frequently accessed data
    extern __shared__ float shared[];

    while (i < n) {
        int xc = i % nx;
        int sc = i / nx;
        int x1 = xcol[xc] - 1, x2 = xcol[xc + 1] - 1;
        int s1 = scol[sc] - 1, s2 = scol[sc + 1] - 1;
        double dd = 0.0;

        while (x1 < x2 || s1 < s2) {
            int xr = (x1 < x2) ? xrow[x1] : INT_MAX;
            int sr = (s1 < s2) ? srow[s1] : INT_MAX;

            // Load values into shared memory
            shared[threadIdx.x] = (sr < xr) ? sval[s1++] : (xr < sr) ? xval[x1++] : (xval[x1++] - sval[s1++]);
            __syncthreads(); // Ensure all threads load their values

            // Use the shared memory data
            double d = shared[threadIdx.x];
            dd += d * d;

            __syncthreads(); // Synchronize before continuing loop
        }

        k[i] = exp(-g * dd);
        i += blockDim.x * gridDim.x;
    }
}