#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _adagrad(int n, float eps, float *dw2, float *dw) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Optimize: Calculate stride once to reduce repeated computation
    while (i < n) {
        float dwi = dw[i]; // Optimize: Load dw[i] into a register to reduce global memory access
        float dwi2 = dwi * dwi; // Optimize: Compute square and reuse value
        dw2[i] += dwi2;
        dw[i] = dwi / (eps + sqrt(dw2[i])); // Optimize: Reuse dwi for division
        i += stride;
    }
}