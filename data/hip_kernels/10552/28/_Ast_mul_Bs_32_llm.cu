#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _Ast_mul_Bs_32(int nx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int gridStride = blockDim.x * gridDim.x;  // Calculate grid stride once

    // Process multiple elements per thread
    for (; i < nx * ns; i += gridStride) {
        double ki = 0.0;
        int xc = i % nx;
        int sc = i / nx;
        int x1 = xcol[xc] - 1, x2 = xcol[xc + 1] - 1;
        int s1 = scol[sc] - 1, s2 = scol[sc + 1] - 1;

        // Unroll inner loop for better performance
        while (x1 < x2 && s1 < s2) {
            int xr = xrow[x1];
            int sr = srow[s1];
            if (sr < xr)
                s1++;
            else if (xr < sr)
                x1++;
            else {
                ki += xval[x1] * sval[s1];
                x1++;
                s1++;
            }
        }
        k[i] = ki;
    }
}