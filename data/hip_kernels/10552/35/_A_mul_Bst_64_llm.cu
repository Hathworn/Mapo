#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _A_mul_Bst_64(int my, int xc, double *dy, double *xval, int *xrow, int *xcol, double *dw) {
    int t = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Compute stride
    int n = xcol[xc + 1] - xcol[xc];
    
    // Loop unrolling for improved performance
    for (int i = t; i < n; i += stride) {
        int xp = xcol[xc] + i;
        int xr = xrow[xp] - 1;

        for (int yr = 0; yr < my; yr++) {
            int yp = yr + xc * my;
            int wp = yr + xr * my;
            atomicAdd(&dw[wp], dy[yp] * xval[xp]); // Use atomicAdd for concurrent updates
        }
    }
}
