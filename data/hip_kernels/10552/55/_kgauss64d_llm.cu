#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _kgauss64d(int nx, int ns, int nd, double *x, double *s, double *k, double g) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int n = nx * ns;

    // Use shared memory to avoid redundant global memory loads
    extern __shared__ double shared_mem[];
    double *shared_s = shared_mem;

    while (i < n) {
        int xj = (i % nx) * nd;
        int sj_base = (i / nx) * nd;

        // Load `s` values into shared memory
        for (int j = 0; j < nd; j++) {
            shared_s[threadIdx.x * nd + j] = s[sj_base + j];
        }
        __syncthreads();

        double dd = 0.0;
        for (int j = 0; j < nd; j++) {
            double d = x[xj + j] - shared_s[threadIdx.x * nd + j];
            dd += d * d;
        }

        k[i] = exp(-g * dd);
        i += blockDim.x * gridDim.x;
        __syncthreads();
    }
}