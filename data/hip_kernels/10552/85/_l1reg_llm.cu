#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _l1reg(int n, float l1, float *w, float *dw) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride once instead of during each loop iteration
    while (i < n) {
        // Use ternary operator to simplify condition checks and updates
        dw[i] += (w[i] > 0) ? l1 : ((w[i] < 0) ? -l1 : 0);
        i += stride;
    }
}