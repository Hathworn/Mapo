#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _kgauss64sum(int xrows, int xcols, double *x, double *xx) {
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride to simplify index calculation

    while (j < xcols) {
        int x0 = j * xrows;
        int x1 = x0 + xrows;
        double sum = 0.0;

        for (int i = x0; i < x1; i++) {
            sum += x[i] * x[i]; // Accumulate squared values
        }
        
        xx[j] = sum; // Write the result to output
        j += stride; // Move to the next column
    }
}