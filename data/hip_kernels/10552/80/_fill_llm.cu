#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _fill(int n, float val, float *x) {
    // Calculate the global index for the current thread
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Unroll the loop to improve performance
    int stride = blockDim.x * gridDim.x;
    for (; i < n; i += stride) {
        x[i] = val;
    }
}