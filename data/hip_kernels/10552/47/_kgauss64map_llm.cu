#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _kgauss64map(int nx, int ns, double *x2, double *s2, double *k, double g) {
    // Calculate global thread index
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int n = nx * ns;

    // Use a stride loop for the grid
    while (i < n) {
        int xi = (i % nx);
        int si = (i / nx);
        
        // Simplified expression inside exp()
        double temp = x2[xi] + s2[si] - 2 * k[i];
        k[i] = exp(-g * temp);

        // Advance to the next grid-stride position
        i += blockDim.x * gridDim.x;
    }
}