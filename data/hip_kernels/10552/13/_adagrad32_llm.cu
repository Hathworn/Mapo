#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _adagrad32(int n, double eps, float *dw2, float *dw) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride outside loop
    while (i < n) {
        float temp_dw = dw[i]; // Cache dw[i] to a register to reduce global memory access
        dw2[i] += temp_dw * temp_dw;
        dw[i] = temp_dw / sqrt(dw2[i] + eps);
        i += stride;
    }
}