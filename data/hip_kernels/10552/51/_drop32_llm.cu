#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _drop32(int n, float *x, float *xmask, float dropout, float scale) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Loop unrolling for better performance
    for (; i < n; i += stride * 4) {
        if (i < n) {
            if (xmask[i] < dropout) x[i] = 0;
            else x[i] *= scale;
        }
        if (i + stride < n) {
            if (xmask[i + stride] < dropout) x[i + stride] = 0;
            else x[i + stride] *= scale;
        }
        if (i + 2 * stride < n) {
            if (xmask[i + 2 * stride] < dropout) x[i + 2 * stride] = 0;
            else x[i + 2 * stride] *= scale;
        }
        if (i + 3 * stride < n) {
            if (xmask[i + 3 * stride] < dropout) x[i + 3 * stride] = 0;
            else x[i + 3 * stride] *= scale;
        }
    }
}