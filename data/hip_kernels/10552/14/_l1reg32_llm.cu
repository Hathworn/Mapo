#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _l1reg32(int n, double l1, float *w, float *dw) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride once to avoid recomputation
    for (; i < n; i += stride) { // Use for loop for better readability
        float wi = w[i]; // Cache value to reduce global memory access
        if (wi > 0) {
            dw[i] += l1;
        } else if (wi < 0) {
            dw[i] -= l1;
        }
    }
}