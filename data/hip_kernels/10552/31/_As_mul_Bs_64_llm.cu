#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _As_mul_Bs_64(int mx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k) {
    int sc = threadIdx.x + blockIdx.x * blockDim.x;
    while (sc < ns) { 
        int k0 = mx * sc;    
        int k1 = k0 + mx;

        // Use memset to initialize k[:,sc] to zero
        if (threadIdx.x == 0) { 
            memset(&k[k0], 0, mx * sizeof(double));
        }
        __syncthreads();  // Ensure all threads sync after memset

        int s0 = scol[sc] - 1;    
        int s1 = scol[sc + 1] - 1;  

        for (int sp = s0; sp < s1; sp++) {
            int sr = srow[sp] - 1;
            double sv = sval[sp];
            int xc = sr;
            int x0 = xcol[xc] - 1;
            int x1 = xcol[xc + 1] - 1;

            for (int xp = x0; xp < x1; xp++) {
                int xr = xrow[xp] - 1;
                double xv = xval[xp];
                atomicAdd(&k[k0 + xr], xv * sv);  // Use atomicAdd for safe concurrent updates
            }
        }
        sc += blockDim.x * gridDim.x;
    }
}