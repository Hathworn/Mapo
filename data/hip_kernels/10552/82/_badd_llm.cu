#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _badd(int nrows, int ncols, float *y, float *b) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global thread index
    int stride = blockDim.x * gridDim.x; // Determine stride
    for (int i = idx; i < ncols * nrows; i += stride) { // Iterate over array using stride
        y[i] += b[i % nrows]; // Update y array element
    }
}