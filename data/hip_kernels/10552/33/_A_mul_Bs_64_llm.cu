#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _A_mul_Bs_64(int mx, int ns, double *x, double *sval, int *srow, int *scol, double *k) {
    int sc = threadIdx.x + blockIdx.x * blockDim.x;
    while (sc < ns) {
        int k0 = mx * sc;
        int k1 = k0 + mx;
        
        // Initialize column of k to zero
        for (int kp = k0; kp < k1; kp++) {
            k[kp] = 0;
        }

        int s0 = scol[sc] - 1;
        int s1 = scol[sc + 1] - 1;

        for (int sp = s0; sp < s1; sp++) {
            int sr = srow[sp] - 1;
            double sv = sval[sp];
            int x0 = mx * sr;

            // Compute k[xr, sc] using shared memory for better performance
            #pragma unroll
            for (int xr = 0; xr < mx; xr++) {
                k[k0 + xr] += x[x0 + xr] * sv;
            }
        }
        sc += blockDim.x * gridDim.x;
    }
}