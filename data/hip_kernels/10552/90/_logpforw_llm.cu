#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _logpforw(int nrows, int ncols, float *y) {
    // Determine column index and range once
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (col < ncols) {
        int i0 = col * nrows;
        int i1 = i0 + nrows;

        // Initialize for reduction
        float ymax = y[i0];
        // Find the maximum value in the column
        for (int i = i0 + 1; i < i1; i++) {
            ymax = fmaxf(y[i], ymax); // Use fmaxf for potential better performance
        }

        // Subtract the maximum and compute sum of exponents
        float z = 0.0f;
        for (int i = i0; i < i1; i++) {
            y[i] -= ymax;
            z += expf(y[i]); // Use expf for single precision
        }

        // Compute the logarithm of the sum of exponents
        float logz = logf(z); // Use logf for single precision

        // Subtract logz to normalize
        for (int i = i0; i < i1; i++) {
            y[i] -= logz;
        }

        // Move to the next column in the grid stride loop
        col += stride;
    }
}