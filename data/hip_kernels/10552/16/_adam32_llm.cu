#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _adam32(int n, int t, double eps, double b1, double b2, float *fstm, float *scndm, float *dw) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    double b1t_inv = 1.0 / (1 - pow(b1, (double)t)); // Precompute 1 - b1^t
    double b2t_inv = 1.0 / (1 - pow(b2, (double)t)); // Precompute 1 - b2^t
    while (i < n) {
        float fstm_val = fstm[i];
        float dw_val = dw[i];
        // Update the first moment estimate
        fstm_val = b1 * fstm_val + (1 - b1) * dw_val;
        // Update the second moment estimate
        float scndm_val = scndm[i];
        scndm_val = b2 * scndm_val + (1 - b2) * (dw_val * dw_val);
        // Compute the bias-corrected estimates and update the weights
        dw[i] = (fstm_val * b1t_inv) / (sqrt(scndm_val * b2t_inv) + eps);

        // Write updates back to global memory
        fstm[i] = fstm_val;
        scndm[i] = scndm_val;

        i += blockDim.x * gridDim.x;
    }
}