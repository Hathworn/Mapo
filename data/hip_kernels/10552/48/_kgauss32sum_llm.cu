#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _kgauss32sum(int xrows, int xcols, float *x, float *xx) {
    // Calculate thread ID
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread ID is within bounds of xcols
    if (j < xcols) {
        int x0 = j * xrows;
        int x1 = x0 + xrows;
        double sum = 0.0;

        // Unroll loop for better performance
        for (int i = x0; i < x1; i++) {
            sum += x[i] * x[i];
        }
        
        xx[j] = sum;
    }
}