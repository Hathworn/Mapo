#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _As_mul_Bs_32(int mx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k) {
    int sc = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;  // Precompute stride
    while (sc < ns) {
        int k0 = mx * sc;
        int k1 = k0 + mx;

        // Initialize k to zero for column sc
        for (int kp = k0; kp < k1; kp++) 
            k[kp] = 0;

        int s0 = scol[sc] - 1;
        int s1 = scol[sc + 1] - 1;
        for (int sp = s0; sp < s1; sp++) {
            int sr = srow[sp] - 1;
            float sv = sval[sp];
            int xc = sr;
            int x0 = xcol[xc] - 1;
            int x1 = xcol[xc + 1] - 1;

            // Calculate contribution to each k element
            for (int xp = x0; xp < x1; xp++) {
                int xr = xrow[xp] - 1;
                atomicAdd(&k[k0 + xr], xval[xp] * sv);  // Use atomicAdd for concurrent updates
            }
        }
        sc += stride;
    }
}