#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _kgauss32(int mx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float g, float *k) {
    // Use shared memory if necessary and unroll loops for better performance
    extern __shared__ float sharedSval[];

    int sc = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;
    int offset = blockDim.x * gridDim.x;

    while (sc < ns) {  // sc: 0-based column for s
        int s0 = scol[sc] - 1;
        int s1 = scol[sc + 1] - 1;
        
        // Load s values into shared memory for faster access
        for (int sp = s0 + tid; sp < s1; sp += blockDim.x) {
            sharedSval[sp - s0] = sval[sp];
        }
        __syncthreads();

        for (int sp = s0; sp < s1; sp++) {
            int sr = srow[sp] - 1;
            float sv = sharedSval[sp - s0];  // Using shared memory

            int xc = sr;
            int x0 = xcol[xc] - 1;
            int x1 = xcol[xc + 1] - 1;

            for (int xp = x0; xp < x1; xp++) {
                int xr = xrow[xp] - 1;
                float xv = xval[xp];
                float xs = xv - sv;

                atomicAdd(&k[mx * sc + xr], xs * xs);  // Use atomicAdd to prevent race conditions
            }
        }

        __syncthreads();

        // Update k array with exponential function
        for (int kp = tid; kp < mx; kp += blockDim.x) {
            int kIdx = mx * sc + kp;
            k[kIdx] = exp(-g * k[kIdx]);
        }
        sc += offset;
    }
}