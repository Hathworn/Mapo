#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _adam64(int n, int t, double eps, double b1, double b2, double *fstm, double *scndm, double *dw) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // Precompute coefficients outside the loop for efficiency
    double b1_pow_t = pow(b1, (double)t);
    double b2_pow_t = pow(b2, (double)t);
    double one_minus_b1_pow_t = 1 - b1_pow_t;
    double one_minus_b2_pow_t = 1 - b2_pow_t;

    while (i < n) {
        double dw_i = dw[i];
        fstm[i] = b1 * fstm[i] + (1 - b1) * dw_i;
        scndm[i] = b2 * scndm[i] + (1 - b2) * (dw_i * dw_i);
        dw[i] = (fstm[i] / one_minus_b1_pow_t) / (sqrt(scndm[i] / one_minus_b2_pow_t) + eps);

        i += blockDim.x * gridDim.x;
    }
}