#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _softback(int nrows, int ncols, float *y, float *dy) {
    // Using shared memory to optimize access memory speed.
    extern __shared__ float sharedMem[];

    float y0, sum;
    int i0, i1;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    while (col < ncols) {
        i0 = col * nrows;
        i1 = i0 + nrows;
        y0 = -INFINITY;
        
        // Load data to shared memory to reduce global memory access.
        for (int i = threadIdx.x; i < nrows; i += blockDim.x) {
            sharedMem[i] = y[i0 + i];
        }
        __syncthreads();

        // Find max value for numerical stability (in shared memory).
        for (int i = 0; i < nrows; i++) {
            if (sharedMem[i] > y0) {
                y0 = sharedMem[i];
            }
        }

        sum = 0;
        // Compute exponential scores and sum (in shared memory).
        for (int i = threadIdx.x; i < nrows; i += blockDim.x) {
            sharedMem[i] = exp(sharedMem[i] - y0);
            sum += sharedMem[i];
        }

        // Use block-level reduction to sum all values.
        __syncthreads();
        float blockSum = atomicAdd(&sum, 0.0f);
        __syncthreads();

        // Normalize probabilities and compute gradient (in shared memory).
        for (int i = threadIdx.x; i < nrows; i += blockDim.x) {
            sharedMem[i] /= blockSum;
            dy[i0 + i] = (sharedMem[i] - dy[i0 + i]) / ncols;
            y[i0 + i] = sharedMem[i];
        }

        col += blockDim.x * gridDim.x;
    }
}