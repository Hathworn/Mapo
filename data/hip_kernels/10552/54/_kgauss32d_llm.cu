#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _kgauss32d(int nx, int ns, int nd, float *x, float *s, float *k, float g) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int n = nx * ns;

    // Using shared memory to reduce global memory access
    extern __shared__ float shared_s[];
    if(threadIdx.x < nd) {
        shared_s[threadIdx.x] = s[threadIdx.x];
    }
    __syncthreads();

    while (i < n) {
        int xj = (i % nx) * nd;
        int sj = (i / nx) * nd;
        double dd = 0;

        // Use local registers to hold values to minimize memory access latency
        for (int j = 0; j < nd; j++) {
            float x_val = x[xj++];
            float s_val = shared_s[j];
            double d = x_val - s_val;
            dd += d * d;
        }
        
        k[i] = exp(-g * dd);
        i += blockDim.x * gridDim.x;
    }
}