#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _l1reg64(int n, double l1, double *w, double *dw) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;  // Calculate stride outside the loop
    while (i < n) {
        double wi = w[i];  // Cache w[i] to reduce global memory access
        if (wi > 0) dw[i] += l1;
        else if (wi < 0) dw[i] -= l1;
        i += stride;
    }
}