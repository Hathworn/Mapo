#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _drop64(int n, double *x, double *xmask, double dropout, double scale) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {  // Use a loop with built-in stride calculation
        if (xmask[i] < dropout) {
            x[i] = 0;
        } else {
            x[i] *= scale;
        }
    }
}