#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _Ast_mul_Bs_64(int nx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Precompute stride to reduce repeated calculations
    int n = nx * ns;
    
    while (i < n) {
        double ki = 0;
        int xc = i % nx;
        int sc = i / nx;
        int x1 = xcol[xc] - 1;
        int x2 = xcol[xc + 1] - 1;
        int s1 = scol[sc] - 1;
        int s2 = scol[sc + 1] - 1;
        
        while (x1 < x2 && s1 < s2) {
            int xr = xrow[x1];
            int sr = srow[s1];
            
            if (sr < xr) {
                s1++;
            } else if (xr < sr) {
                x1++;
            } else {
                ki += xval[x1++] * sval[s1++]; // Eliminate braces for single line conditions
            }
        }
        k[i] = ki;
        i += stride; // Use precomputed stride
    }
}