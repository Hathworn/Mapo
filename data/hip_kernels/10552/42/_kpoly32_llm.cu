#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _kpoly32(int nx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k, float c, float d) {
    // Compute global index
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int n = nx*ns;

    // Loop for processing multiple elements per thread with stride
    for (; i < n; i += blockDim.x * gridDim.x) {
        double ki = 0;
        // Calculate column indices for x and s
        int xc = i % nx;
        int sc = i / nx;
        // Set row index boundaries for x and s
        int x1 = xcol[xc] - 1;
        int x2 = xcol[xc + 1] - 1;
        int s1 = scol[sc] - 1;
        int s2 = scol[sc + 1] - 1;

        // Loop to process current element's column until either x or s is exhausted
        while (x1 < x2 && s1 < s2) {
            int xr = xrow[x1];
            int sr = srow[s1];
            // Compare and update indices based on row comparison for x and s
            if (sr < xr) {
                s1++;
            } else if (xr < sr) {
                x1++;
            } else {
                // Accumulate product of matching elements
                ki += xval[x1++] * sval[s1++];
            }
        }
        // Compute kernel value using polynomial expansion
        k[i] = pow(ki + c, (double)d);
    }
}