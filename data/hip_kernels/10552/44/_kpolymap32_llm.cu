#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _kpolymap32(int n, float *k, float c, float d) {
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for better occupancy
    int stride = blockDim.x * gridDim.x;
    for (; i < n; i += stride) {
        k[i] = powf(k[i] + c, d); // Use powf for single precision
    }
}