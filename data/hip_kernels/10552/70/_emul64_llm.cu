#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _emul64(int n, double *x, double *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Optimize by using stride-based loop
    for (; i < n; i += stride) {
        y[i] *= x[i]; // Perform in-place multiplication
    }
}