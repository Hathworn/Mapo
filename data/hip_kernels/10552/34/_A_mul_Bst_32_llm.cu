#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _A_mul_Bst_32(int my, int xc, float *dy, float *xval, int *xrow, int *xcol, float *dw) {
    // Calculate global thread ID
    int t = threadIdx.x + blockIdx.x * blockDim.x;

    // Compute number of elements in current column
    int n = xcol[xc+1] - xcol[xc];

    // Efficiently handle threads within bounds
    for (int idx = t; idx < n; idx += blockDim.x * gridDim.x) {
        int xp = xcol[xc] + idx - 1;
        int xr = xrow[xp] - 1;

        // Optimize loop unrolling for my
        for (int yr = 0; yr < my; ++yr) {
            int yp = yr + xc * my;
            int wp = yr + xr * my;
            atomicAdd(&dw[wp], dy[yp] * xval[xp]);  // Use atomicAdd for concurrent writes
        }
    }
}