#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _drop32(int n, float *x, float *y, float *xmask, double dropout, double scale) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Precompute stride for all threads
    while (i < n) {
        // Utilize ternary operator for conditional assignments
        y[i] = (xmask[i] < dropout) ? 0.0f : x[i] * scale;
        i += stride;
    }
}