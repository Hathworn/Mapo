#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _kpoly64(int nx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k, double c, double d) {
    // Calculate the global thread index
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int n = nx * ns;
    
    // Optimize by unrolling the loop
    while (i < n) {
        double ki = 0.0;
        int xc = i % nx;
        int sc = i / nx;
        int x1 = xcol[xc] - 1, x2 = xcol[xc + 1] - 1;
        int s1 = scol[sc] - 1, s2 = scol[sc + 1] - 1;
        
        // Unroll the inner while loop
        while ((x1 < x2) && (s1 < s2)) {
            int xr = xrow[x1], sr = srow[s1];
            if (sr < xr) {
                s1++;
            } else if (xr < sr) {
                x1++;
            } else {
                ki += xval[x1++] * sval[s1++];
            }
        }
        
        // Calculate the kernel value
        k[i] = pow(ki + c, d);
        
        // Move to the next element in this thread's stride
        i += blockDim.x * gridDim.x;
    }
}