#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _A_mul_Bs_32(int mx, int ns, float *x, float *sval, int *srow, int *scol, float *k) {
    int sc = threadIdx.x + blockIdx.x * blockDim.x;

    // Pre-calculate the stride to avoid recalculation in while loop
    int stride = blockDim.x * gridDim.x;

    while (sc < ns) {
        int k0 = mx * sc;      // Start index for k[:,sc]
        int s0 = scol[sc] - 1; // Start index for s[:,sc]
        int s1 = scol[sc + 1] - 1; // End index for s[:,sc], exclusive

        // Initialize k[:,sc] to zero more efficiently
        for (int kp = 0; kp < mx; ++kp) {
            k[k0 + kp] = 0.0f;
        }

        // Perform the sparse matrix-vector multiplication
        for (int sp = s0; sp < s1; ++sp) {
            int sr = srow[sp] - 1; // Convert to 0-based indexing
            float sv = sval[sp];
            int x0 = mx * sr; // Start index for x[:,sr]

            // Perform the vector operations inside the loop
            for (int xr = 0; xr < mx; ++xr) {
                float xv = x[x0 + xr];
                k[k0 + xr] += xv * sv;
            }
        }

        // Move to the next sc index using pre-calculated stride
        sc += stride;
    }
}