#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _reluback(int n, float *y, float *dy) {
    // Use stride to cover the entire range
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += blockDim.x * gridDim.x) {
        // Set gradient to zero for inactive ReLU neurons
        dy[i] = (y[i] <= 0) ? 0 : dy[i];
    }
}