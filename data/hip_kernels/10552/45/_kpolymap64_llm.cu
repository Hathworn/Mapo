#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _kpolymap64(int n, double *k, double c, double d) {
    // Use shared memory for storing computation results
    extern __shared__ double sharedMem[];

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // Loop unrolling optimization
    while (i < n) {
        sharedMem[threadIdx.x] = pow(k[i] + c, d);

        // Store computed value back to global memory
        k[i] = sharedMem[threadIdx.x];

        i += blockDim.x * gridDim.x;
    }
}