#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _rmsprop64(int n, double eps, double rho, double *dw2, double *dw) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Precompute stride to minimize repeated calculations
    if (i < n) { // Ensure index is within bounds
        double local_eps = eps; // Use register for frequently accessed variables
        double local_rho = rho;
        for (int idx = i; idx < n; idx += stride) { // Use for-loop to replace while-loop for readability
            double dw_val = dw[idx];
            dw2[idx] = dw2[idx] * local_rho + (1.0 - local_rho) * dw_val * dw_val;
            dw[idx] /= sqrt(dw2[idx] + local_eps);
        }
    }
}