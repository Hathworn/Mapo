#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _adagrad64(int n, double eps, double *dw2, double *dw) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Calculate the stride size once

    for (; i < n; i += stride) { // Use a for loop with pre-calculated stride
        double dw_val = dw[i];
        double dw2_val = dw2[i];
        dw2_val += dw_val * dw_val; // Cache access in registers
        dw2[i] = dw2_val;
        dw[i] = dw_val / sqrt(dw2_val + eps); // Use cached value
    }
}