#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _logploss(int nrows, int ncols, float *y, float *dy) {
    // Calculate unique index for each thread
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Efficient stride to cover elements by threads
    int totalThreads = blockDim.x * gridDim.x;
    
    for (int col = tid; col < ncols; col += totalThreads) {
        int i0 = col * nrows;
        int i1 = i0 + nrows;
        
        // Process each row for current column
        for (int i = i0; i < i1; i++) {
            dy[i] = (expf(y[i]) - dy[i]) / ncols;  // Use expf for better performance with float
        }
    }
}