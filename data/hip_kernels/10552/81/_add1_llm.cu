#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _add1(int n, float val, float *x) {
    // Calculate global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;

    // Use grid stride loop for better utilization of threads
    for (; i < n; i += gridSize) {
        x[i] += val;
    }
}