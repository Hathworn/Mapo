#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _drop64(int n, double *x, double *y, double *xmask, double dropout, double scale) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (; i < n; i += stride) { // Unrolling loop and using stride for better performance
        double mask_val = xmask[i]; // Reduce global memory access
        y[i] = (mask_val < dropout) ? 0 : x[i] * scale; // Use ternary operator
    }
}