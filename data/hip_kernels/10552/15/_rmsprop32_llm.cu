#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _rmsprop32(int n, double eps, double rho, float *dw2, float *dw) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    // Use a loop stride for efficient parallelization
    for (; i < n; i += stride) {
        // Calculate updated dw2 and apply RMSProp formula
        float dw_temp = dw[i]; // Cache access to dw[i]
        dw2[i] = dw2[i] * rho + (1 - rho) * dw_temp * dw_temp;
        dw[i] = dw_temp / sqrtf(dw2[i] + eps); // Use sqrtf for float precision
    }
}