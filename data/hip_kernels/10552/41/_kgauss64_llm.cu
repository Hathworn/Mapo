#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _kgauss64(int nx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k, double g) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int n = nx * ns;

    // Iterate over i and compute Gaussian kernel
    while (i < n) {
        int xc = i % nx;
        int sc = i / nx;
        int x1 = xcol[xc] - 1, x2 = xcol[xc + 1] - 1;
        int s1 = scol[sc] - 1, s2 = scol[sc + 1] - 1;
        double dd = 0;

        // Improve loop efficiency by unrolling
        while (x1 < x2 && s1 < s2) {
            int xr = xrow[x1];
            int sr = srow[s1];

            if (xr == sr) {
                double d = xval[x1++] - sval[s1++];
                dd += d * d;
            } else if (xr < sr) {
                double d = xval[x1++];
                dd += d * d;
            } else {
                double d = sval[s1++];
                dd += d * d;
            }
        }

        while (x1 < x2) {
            double d = xval[x1++];
            dd += d * d;
        }
        
        while (s1 < s2) {
            double d = sval[s1++];
            dd += d * d;
        }

        // Compute Gaussian value
        k[i] = exp(-g * dd);

        // Advance to next iteration
        i += blockDim.x * gridDim.x;
    }
}