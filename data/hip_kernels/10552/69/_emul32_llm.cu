#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _emul32(int n, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    int gridSize = blockDim.x * gridDim.x; // Calculate total number of threads in the grid
    for (; i < n; i += gridSize) { // Use for loop for better readability
        y[i] *= x[i];
    }
}