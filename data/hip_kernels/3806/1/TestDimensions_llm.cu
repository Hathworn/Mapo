#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void TestDimensions()
{
    // Pre-calculate common products to reduce redundant calculations
    int const num_threads_per_block = blockDim.x * blockDim.y * blockDim.z;
    int const num_blocks = gridDim.x * gridDim.y * gridDim.z;
    int const threads_stride = num_threads_per_block * num_blocks;

    int const blockIdx_xy = blockIdx.x + blockIdx.y * gridDim.x;
    int const blockIdx_xyz = blockIdx_xy + blockIdx.z * gridDim.x * gridDim.y;
    int const threadIdx_xy = threadIdx.x + threadIdx.y * blockDim.x;

    // Calculate unique thread ID
    int const thread_id = blockIdx_xyz * num_threads_per_block + threadIdx_xy + threadIdx.z * blockDim.x * blockDim.y;

    if (thread_id == 0)
    {
        printf("gridDim   = x: %6d / y: %6d / z: %6d\r\n", gridDim.x, gridDim.y, gridDim.z);
        printf("blockDim  = x: %6d / y: %6d / z: %6d\r\n", blockDim.x, blockDim.y, blockDim.z);
        printf("num_threads_per_block: %6d\r\n", num_threads_per_block);
        printf("num_blocks           : %6d\r\n", num_blocks);
        printf("threads_stride       : %6d\r\n", threads_stride);
    }

    // Print thread and block details
    printf("tidx | %6d | %6d | %6d | bidx | %6d | %6d | %6d | "
           "gdim | %6d | %6d | %6d | bdim | %6d | %6d | %6d | "
           "thread_id | %6d |\r\n",
           threadIdx.x, threadIdx.y, threadIdx.z,
           blockIdx.x, blockIdx.y, blockIdx.z,
           gridDim.x, gridDim.y, gridDim.z,
           blockDim.x, blockDim.y, blockDim.z, thread_id);

    return;
}