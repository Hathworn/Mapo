#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Determine the thread unique index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure only one thread performs the division
    if(idx == 0) {
        *accuracy /= N;
    }
}