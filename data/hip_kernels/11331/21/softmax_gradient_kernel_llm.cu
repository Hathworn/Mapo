#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SOFTMAX_NUM_THREADS 256

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0;

    // Unroll loop for better parallel execution
    for (int i = idx; i < dim; i += blockDim.x * 4) {
        tmp += dY[i] * Y[i];
        if (i + blockDim.x < dim) tmp += dY[i + blockDim.x] * Y[i + blockDim.x];
        if (i + 2 * blockDim.x < dim) tmp += dY[i + 2 * blockDim.x] * Y[i + 2 * blockDim.x];
        if (i + 3 * blockDim.x < dim) tmp += dY[i + 3 * blockDim.x] * Y[i + 3 * blockDim.x];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Optimize reduction with atomic addition
    if (idx == 0) {
        tmp = reduction_buffer[0];
        for (int i = 1; i < blockDim.x; ++i) {
            atomicAdd(&tmp, reduction_buffer[i]);
        }
        reduction_buffer[0] = tmp;
    }
    __syncthreads();

    // Unroll loop for gradient computation
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x * 4) {
        dX[i] = Y[i] * (dY[i] - tmp);
        if (i + blockDim.x < dim) dX[i + blockDim.x] = Y[i + blockDim.x] * (dY[i + blockDim.x] - tmp);
        if (i + 2 * blockDim.x < dim) dX[i + 2 * blockDim.x] = Y[i + 2 * blockDim.x] * (dY[i + 2 * blockDim.x] - tmp);
        if (i + 3 * blockDim.x < dim) dX[i + 3 * blockDim.x] = Y[i + 3 * blockDim.x] * (dY[i + 3 * blockDim.x] - tmp);
    }
}