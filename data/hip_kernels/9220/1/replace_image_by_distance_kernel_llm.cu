#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void replace_image_by_distance_kernel(const unsigned char *pImage, const float* pDepth, const unsigned char *pBackground, unsigned char *result, const float max_value, const unsigned int width, const unsigned int height, const unsigned int image_channels)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Early exit for threads outside image bounds
    if (y >= height || x >= width)
        return;

    // Calculate indices once
    const int img_index = (y * width + x) * image_channels;
    const int result_index = (y * width + x) * 3;

    // Get depth and determine pixel replacement
    float z_distance = pDepth[y * width + x];
    if (isfinite(z_distance) && (z_distance > max_value))
    {
        result[result_index] = pBackground[result_index];
        result[result_index + 1] = pBackground[result_index + 1];
        result[result_index + 2] = pBackground[result_index + 2];
    }
    else
    {
        // Handle both gray and color images
        result[result_index] = pImage[img_index];
        result[result_index + 1] = (image_channels == 1) ? pImage[y * width + x] : pImage[img_index + 1];
        result[result_index + 2] = (image_channels == 1) ? pImage[y * width + x] : pImage[img_index + 2];
    }
}