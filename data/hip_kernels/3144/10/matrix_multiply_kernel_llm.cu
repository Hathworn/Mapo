#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < dim_mn) {
        double out = 0.0;
        // Unrolling the loop for better performance
        for (long j = 0; j < dim_mn; j+=4) {
            out += matrix[i * dim_mn + j] * vector_in[j];
            if (j+1 < dim_mn) out += matrix[i * dim_mn + j + 1] * vector_in[j + 1];
            if (j+2 < dim_mn) out += matrix[i * dim_mn + j + 2] * vector_in[j + 2];
            if (j+3 < dim_mn) out += matrix[i * dim_mn + j + 3] * vector_in[j + 3];
        }
        vector_out[i] = out;
    }
}