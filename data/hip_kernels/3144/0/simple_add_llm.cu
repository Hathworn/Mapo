#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_add(float* __restrict__ a, const float* __restrict__ b, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        a[i] += b[i]; // Use shorthand operator
    }
}