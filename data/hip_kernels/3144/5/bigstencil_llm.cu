#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bigstencil(int* in, int* out) {
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory to optimize memory access
    __shared__ int sharedMem[256];
    
    // Load elements into shared memory
    sharedMem[threadIdx.x] = in[i];
    __syncthreads();
    
    // Perform computation using shared memory
    out[i] = sharedMem[threadIdx.x] + 2;
}