#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction(int* input, int* output) {
    __shared__ int tmp[TPB];

    // Each thread loads one element from global to shared memory
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    tmp[threadIdx.x] = input[idx];

    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            tmp[threadIdx.x] += tmp[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        output[blockIdx.x] = tmp[0];
    }
}