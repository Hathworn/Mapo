#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate bottom left corner of the target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    
    // Use modff for finding pixel coords
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    
    // Pre-compute weights
    float dx1 = 1.0f - dx;
    float dy1 = 1.0f - dy;
    
    // Target pixel integer coords and value
    int tx = (int) px;
    int ty = (int) py;
    float value = src[image_row_offset + j];

    // Consolidate bounds check and atomic adds
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0)) {
        float weight = dx * dy;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    tx -= 1;
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0)) {
        float weight = dx1 * dy;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    ty -= 1;
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0)) {
        float weight = dx1 * dy1;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    tx += 1;
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0)) {
        float weight = dx * dy1;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}