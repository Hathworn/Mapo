#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeLocation(const short2* __restrict__ loc_, float* __restrict__ x, float* __restrict__ y, const int npoints, const float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use if condition only if necessary to minimize divergence
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}