#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Compute global thread index
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (i >= h || j >= w) return;

    const int pos = i * s + j;

    float scale = normalization_factor[pos];
    float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

    // Normalize image
    image[pos] *= invScale;
}