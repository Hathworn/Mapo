#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Round(float * A, float * out, int size) {
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    if (id < size) {
        out[id] = roundf(out[id]);  // Use roundf for potential better precision and performance.
    }
}