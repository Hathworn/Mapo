#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void LengthFromElements(float* element1, float* element2, float* output, int count)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation

    if (threadId < count)
    {
        float e1 = element1[threadId];
        float e2 = element2[threadId];
        output[threadId] = sqrtf(e1 * e1 + e2 * e2); // Avoid repeated array access
    }
}