#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DrawMaskedColorKernel2DBlock(float *target, int targetWidth, int targetHeight, int inputX, int inputY, float *textureMask, int textureWidth, int textureHeight, float r, float g, float b)
{
    int idTextureRgb = blockIdx.y; // Color channel index
    int idTextureY = blockIdx.x * blockDim.y + threadIdx.y; // Y coordinate in texture
    int idTextureX = threadIdx.x; // X coordinate in texture

    if (idTextureRgb < 3) // Only RGB channels are interesting
    {
        // if the texture pixel offset by inputX, inputY, lies inside the target
        if (idTextureX + inputX < targetWidth &&
            idTextureX + inputX >= 0 &&
            idTextureY + inputY < targetHeight &&
            idTextureY + inputY >= 0)
        {
            int tIndex = targetWidth * targetHeight * idTextureRgb + targetWidth * (idTextureY + inputY) + (idTextureX + inputX);
            int aIndex = idTextureY * textureWidth + idTextureX + 3 * textureWidth * textureHeight; // Optimized alpha index calculation
            float a = textureMask[aIndex];

            if (a > 0) // Mask allows color here
            {
                switch (idTextureRgb)
                {
                    case 0:
                        target[tIndex] = r;
                        break;
                    case 1:
                        target[tIndex] = g;
                        break;
                    case 2:
                        target[tIndex] = b;
                        break;
                }
            }
        }
    }
}