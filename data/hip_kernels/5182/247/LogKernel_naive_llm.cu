#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LogKernel_optimized(const float *A, int Acount, int Acols, float *out0, int out0count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    
    // Check bounds for valid threads only
    if (idx < out0count)
    {
        out0[idx] = logf(A[idx]); // Use logf for faster computation with float
    }
}