#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void IntervalToBinaryVector(float input, float* outputs, int steps)
{
    // Calculate unique global thread ID directly
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Use if statement only if id is within range
    if (id < steps)
    {
        // Precompute fraction multiplication for efficiency
        float fraction = 1.0f / steps;
        float lower_bound = fraction * id;
        float upper_bound = fraction * (id + 1);

        // Store result in outputs array
        outputs[id] = (input >= lower_bound && input <= upper_bound);
    }
}