#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using 2D grid and block index calculations
__global__ void Multiply(float* im, float val, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * gridDim.x * blockDim.x);
    if (id < size)
    {
        im[id] *= val;
    }
}