#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FullyConnectedAdjustMemoryKernel( float *weightsGradPtr, float *biasGradPtr, float *weightGradCurvePtr, float *biasGradCurvePtr, float *avgWeightGradPtr, float *avgBiasGradPtr, float *avgWeightGradVarPtr, float *avgBiasGradVarPtr, float *avgWeightGradCurvePtr, float *avgBiasGradCurvePtr, float *avgWeightGradCurveVarPtr, float *avgBiasGradCurveVarPtr, float *weightMemorySizePtr, float *biasMemorySizePtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize )
{
    int j = blockDim.x * blockIdx.y * gridDim.x + blockDim.x * blockIdx.x + threadIdx.x;

    if (j < thisLayerSize && !dropoutMaskPtr[j])
    {
        int index = j;
        
        float avgWeightGrad = avgWeightGradPtr[index];
        float avgWeightGradVar = avgWeightGradVarPtr[index];
        float avgWeightGradCurve = avgWeightGradCurvePtr[index];
        float avgWeightGradCurveVar = avgWeightGradCurveVarPtr[index];

        float weightThreshold = 2 * sqrtf(avgWeightGradVar) - avgWeightGrad * avgWeightGrad;
        float weightCurveThreshold = 2 * sqrtf(avgWeightGradCurveVar - avgWeightGradCurveVar * avgWeightGradCurveVar);

        for (int i = 0; i < prevLayerSize; i++)
        {
            // Use pre-calculated threshold for efficiency
            if (abs(weightsGradPtr[index] - avgWeightGrad) > weightThreshold ||
                abs(weightGradCurvePtr[index] - avgWeightGradCurve) > weightCurveThreshold)
            {
                weightMemorySizePtr[index] = 2.2f;
            }
            index += thisLayerSize;
        }

        float avgBiasGrad = avgBiasGradPtr[j];
        float avgBiasGradVar = avgBiasGradVarPtr[j];
        float avgBiasGradCurve = avgBiasGradCurvePtr[j];
        float avgBiasGradCurveVar = avgBiasGradCurveVarPtr[j];

        float biasThreshold = 2 * sqrtf(avgBiasGradVar) - avgBiasGrad * avgBiasGrad;
        float biasCurveThreshold = 2 * sqrtf(avgBiasGradCurveVar - avgBiasGradCurveVar * avgBiasGradCurveVar);

        // Use pre-calculated threshold for efficiency
        if (abs(biasGradPtr[j] - avgBiasGrad) > biasThreshold ||
            abs(biasGradCurvePtr[j] - avgBiasGradCurve) > biasCurveThreshold)
        {
            biasMemorySizePtr[j] = 2.2f;
        }
    }
}