#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void crate3Dplot(float* plotValues, float patchSize, int itemsX, int itemsY, float maxValue, float* vertexData)
{
    int threadId = blockDim.x * blockIdx.y * gridDim.x  // rows preceding current row in grid
                 + blockDim.x * blockIdx.x              // blocks preceding current block
                 + threadIdx.x;

    int size = itemsX * itemsY;
    float texSizeX = 1.0f / itemsX;
    float texSizeY = 1.0f / itemsY;

    int patchX = threadId / itemsY;
    int patchY = itemsY - (threadId % itemsY) - 1;

    if (threadId < size)
    {
        float height = plotValues[threadId] / maxValue;
        float gap = 0.000;

        float3* vertTop = (float3*)vertexData;
        float2* texCoords = (float2*)(vertexData + (60 * size));

        // Pre-calculate indices for efficiency
        int baseIdx = threadId * 4;
        
        // top side
        float x1 = patchX * patchSize + gap;
        float x2 = (patchX + 1) * patchSize - gap;
        float y1 = patchY * patchSize + gap;
        float y2 = (patchY + 1) * patchSize - gap;
        
        vertTop[baseIdx]     = {x1, height, y1};
        vertTop[baseIdx + 1] = {x2, height, y1};
        vertTop[baseIdx + 2] = {x2, height, y2};
        vertTop[baseIdx + 3] = {x1, height, y2};

        texCoords[baseIdx]     = {patchX * texSizeX, patchY * texSizeY};
        texCoords[baseIdx + 1] = {(patchX + 1) * texSizeX, patchY * texSizeY};
        texCoords[baseIdx + 2] = {(patchX + 1) * texSizeX, (patchY + 1) * texSizeY};
        texCoords[baseIdx + 3] = {patchX * texSizeX, (patchY + 1) * texSizeY};

        float3* vertLeft = (float3*)(vertexData + 12 * size);
        float3* vertFar = (float3*)(vertexData + 24 * size);
        float3* vertNear = (float3*)(vertexData + 36 * size);
        float3* vertRight = (float3*)(vertexData + 48 * size);

        // Simplify vertice generation
        #define SET_VERTICES(vertArray, v0, v1, v2, v3) \
            vertArray[baseIdx]     = v0; vertArray[baseIdx].y = 0; \
            vertArray[baseIdx + 1] = v1; vertArray[baseIdx + 1].y = 0; \
            vertArray[baseIdx + 2] = v2; \
            vertArray[baseIdx + 3] = v3; vertArray[baseIdx + 3].y = 0;

        SET_VERTICES(vertLeft, vertTop[baseIdx], vertTop[baseIdx], vertTop[baseIdx + 3], vertTop[baseIdx + 3])
        SET_VERTICES(vertFar,  vertTop[baseIdx + 2], vertTop[baseIdx + 3], vertTop[baseIdx + 3], vertTop[baseIdx + 2])
        SET_VERTICES(vertNear, vertTop[baseIdx + 1], vertTop[baseIdx], vertTop[baseIdx], vertTop[baseIdx + 1])
        SET_VERTICES(vertRight, vertTop[baseIdx + 2], vertTop[baseIdx + 2], vertTop[baseIdx + 1], vertTop[baseIdx + 1])
    }
}