#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FullyConnectedShiftKernel(float *weightPtr, float *biasPtr, float *shiftedWeightsPtr, float *shiftedBiasPtr, float *avgWeightGradPtr, float *avgBiasGradPtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize)
{
    // j: current layer neuron id
    int j = blockDim.x * blockIdx.y * gridDim.x  // rows preceding current row in grid
            + blockDim.x * blockIdx.x            // blocks preceding current block
            + threadIdx.x;

    if (j < thisLayerSize)
    {
        if (!dropoutMaskPtr[j])
        {
            // Use shared memory for better performance
            extern __shared__ float sharedMem[];
            float *sharedWeights = sharedMem;
            float *sharedBias = (float*)&sharedWeights[prevLayerSize];

            for (int i = 0; i < prevLayerSize; i++)
            {
                // Load to shared memory
                sharedWeights[i] = weightPtr[i * thisLayerSize + j];
            }
            sharedBias[j] = biasPtr[j];

            __syncthreads();

            // weight gradient
            int index = j;
            for (int i = 0; i < prevLayerSize; i++)
            {
                shiftedWeightsPtr[index] = sharedWeights[i] + avgWeightGradPtr[index];
                index += thisLayerSize;
            }

            // bias gradient
            shiftedBiasPtr[j] = sharedBias[j] - avgBiasGradPtr[j]; 
        }
    }
}