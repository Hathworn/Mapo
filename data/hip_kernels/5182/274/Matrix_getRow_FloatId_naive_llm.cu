#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_getRow_FloatId_optimized(const float *A, int Acount, int Acols, float *out0, int out0count, int out0cols, float row_id) {
    // Calculate 1D block index for better grid and block coordination
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < Acols) { 
        // Pre-calculate row start index in global memory to reduce redundant computation
        int row_start_idx = static_cast<int>(row_id) * Acols;
        out0[id] = A[row_start_idx + id];
    }
}