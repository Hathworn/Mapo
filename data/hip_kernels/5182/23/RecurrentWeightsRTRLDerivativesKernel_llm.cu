#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RecurrentWeightsRTRLDerivativesKernel( float *previousHiddenActivations, float *hiddenActivationDerivatives, float *recurrentWeights, float *recurrentWeightRTRLDerivatives, float *previousRecurrentWeightRTRLDerivatives )
{
    int partialId = blockDim.x * blockIdx.y * gridDim.x	// Calculate the unique identifier for each thread
                     + blockDim.x * blockIdx.x
                     + threadIdx.x;

    if (partialId < D_HIDDEN_UNITS * D_HIDDEN_UNITS * D_HIDDEN_UNITS)
    {
        int unitId = partialId / (D_HIDDEN_UNITS * D_HIDDEN_UNITS);
        int weightId = partialId % (D_HIDDEN_UNITS * D_HIDDEN_UNITS);
        int to = weightId / D_HIDDEN_UNITS;
        int from = weightId % D_HIDDEN_UNITS;

        float sum = 0;
        // Use shared memory for inputs to improve cache hits
        __shared__ float sharedRecurrentWeights[D_HIDDEN_UNITS * D_HIDDEN_UNITS];
        __shared__ float sharedPreviousRTRLDerivatives[D_HIDDEN_UNITS * D_HIDDEN_UNITS * D_HIDDEN_UNITS];
        
        // Load elements into shared memory
        if (threadIdx.x < D_HIDDEN_UNITS * D_HIDDEN_UNITS) {
            sharedRecurrentWeights[threadIdx.x] = recurrentWeights[threadIdx.x];
            sharedPreviousRTRLDerivatives[threadIdx.x] = previousRecurrentWeightRTRLDerivatives[threadIdx.x];
        }
        __syncthreads();

        for (int i = 0; i < D_HIDDEN_UNITS; i++)
        {
            sum += sharedRecurrentWeights[unitId * D_HIDDEN_UNITS + i] * sharedPreviousRTRLDerivatives[i * (D_HIDDEN_UNITS * D_HIDDEN_UNITS) + weightId];
        }

        recurrentWeightRTRLDerivatives[partialId] = hiddenActivationDerivatives[unitId] * ((unitId == to) * previousHiddenActivations[from] + sum);
    }
}