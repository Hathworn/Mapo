#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DrawRgbaColorKernel(float *target, int targetWidth, int targetHeight, int inputX, int inputY, int areaWidth, int areaHeight, float r, float g, float b)
{
    int id = blockDim.x * blockIdx.y * gridDim.x
           + blockDim.x * blockIdx.x
           + threadIdx.x;

    int targetPixels = targetWidth * targetHeight;
    int texturePixels = areaWidth * areaHeight;

    int idTextureRgb = id / texturePixels;
    int idTexturePixel = id % texturePixels; // simplified modulus operation
    int idTextureY = idTexturePixel / areaWidth;
    int idTextureX = idTexturePixel % areaWidth;

    if (idTextureRgb < 3) // Optimized condition check
    {
        int posX = idTextureX + inputX; // precompute frequently used value
        int posY = idTextureY + inputY;

        if (posX < targetWidth && posX >= 0 && posY < targetHeight && posY >= 0)
        {
            float color;
            if (idTextureRgb == 0)
                color = r;
            else if (idTextureRgb == 1)
                color = g;
            else
                color = b;

            int tIndex = targetPixels * idTextureRgb + targetWidth * posY + posX;
            target[tIndex] = color;
        }
    }
}