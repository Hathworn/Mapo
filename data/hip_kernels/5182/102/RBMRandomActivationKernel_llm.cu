#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float activateRandomly(float probability, float random)
{
    return random < probability;
}

__global__ void RBMRandomActivationKernel(float *outputPtr, float *randomPtr, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation for 1D grid layout
    
    if (i < size)
    {
        outputPtr[i] = activateRandomly(outputPtr[i], randomPtr[i]);
    }
}