#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DrawRgbaTextureKernelNearestNeighbor(float *target, int targetWidth, int targetHeight, 
                                                    int inputX, int inputY, float *texture, 
                                                    int textureWidth, int textureHeight, 
                                                    int objectWidth, int objectHeight)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Optimize id calculation
    int objectPixels = objectWidth * objectHeight;
    
    int idObjectRgb = id / objectPixels;
    int idObjectPixel = id % objectPixels; // Simplified modulo calculation
    int idObjectY = idObjectPixel / objectWidth;
    int idObjectX = idObjectPixel % objectWidth; // Simplified modulo calculation

    if (idObjectRgb < 3) // 3 channels that we will write to
    {
        int targetRgb = (idObjectRgb == 0) ? 2 : (idObjectRgb == 2 ? 0 : 1); // Inline channel mapping

        if (idObjectX + inputX < targetWidth && idObjectX + inputX >= 0 && 
            idObjectY + inputY < targetHeight && idObjectY + inputY >= 0)
        {
            int textureX = textureWidth * idObjectX / objectWidth; // Direct integer division
            int textureY = textureHeight * idObjectY / objectHeight; // Direct integer division
            int textureId = textureY * textureWidth + textureX;

            float textureValue = texture[textureId + idObjectRgb * (textureWidth * textureHeight)]; // Inline multiplication
            int tIndex = (targetWidth * (idObjectY + inputY) + (idObjectX + inputX)) + idObjectRgb * (targetWidth * targetHeight);

            float a = texture[textureId + 3 * (textureWidth * textureHeight)]; // Inline multiplication
            target[tIndex] = target[tIndex] * (1.0f - a) + a * textureValue;
        }
    }
}