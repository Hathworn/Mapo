#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Crop2DKernel(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int size, int leftMargin, int topMargin, float fillValue)
{
    // Compute global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    if (id < size)
    {
        // Calculate input coordinates
        int inputX = id % outputWidth - leftMargin;
        int inputY = id / outputWidth - topMargin;

        // Check coordinate validity and assign value
        if (inputX >= 0 && inputX < inputWidth && inputY >= 0 && inputY < inputHeight)
            output[id] = input[inputX + inputY * inputWidth];
        else
            output[id] = fillValue;
    }
}