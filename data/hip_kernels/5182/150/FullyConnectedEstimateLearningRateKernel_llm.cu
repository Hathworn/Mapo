#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FullyConnectedEstimateLearningRateKernel(float *weightLearningRatePtr, float *biasLearningRatePtr, float *avgWeightGradPtr, float *avgBiasGradPtr, float *avgWeightGradVarPtr, float *avgBiasGradVarPtr, float *avgWeightGradCurvePtr, float *avgBiasGradCurvePtr, float *avgWeightGradCurveVarPtr, float *avgBiasGradCurveVarPtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize)
{
    int j = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    // Check if j is within bounds and not dropped out
    if (j < thisLayerSize && !dropoutMaskPtr[j])
    {
        float gradCurveVarInv = 1.0f / avgWeightGradCurveVarPtr[j];
        float gradVarInv = 1.0f / avgWeightGradVarPtr[j];
        float avgBiasGradCurve = avgBiasGradCurvePtr[j];
        float avgBiasGrad = avgBiasGradPtr[j];
        float avgBiasGradVar = avgBiasGradVarPtr[j];

        for (int i = 0, index = j; i < prevLayerSize; i++, index += thisLayerSize)
        {
            // Precompute invariant and save redundant calculations
            float weightGradCurve = avgWeightGradCurvePtr[index] * gradCurveVarInv;
            float weightGrad = avgWeightGradPtr[index] * gradVarInv;
            weightLearningRatePtr[index] = weightGradCurve * weightGrad * avgWeightGradPtr[index];
        }

        // Simplify bias learning rate computation
        biasLearningRatePtr[j] = (avgBiasGradCurve / avgBiasGradCurveVarPtr[j]) * (avgBiasGrad * avgBiasGrad / avgBiasGradVar);
    }
}