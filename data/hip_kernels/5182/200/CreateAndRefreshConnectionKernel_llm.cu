#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CreateAndRefreshConnectionKernel(int s1, int s2, int *connection, int *age, int maxCells)
{
    // Calculate unique thread index
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Single-thread optimization to avoid unnecessary calculations
    if (threadId == 0) {
        int index1 = s1 * maxCells + s2;
        int index2 = s2 * maxCells + s1;
        
        connection[index1] = 1;
        age[index1] = 0;
        connection[index2] = 1;
        age[index2] = 0;
    }
}