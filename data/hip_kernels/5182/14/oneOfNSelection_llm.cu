#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void oneOfNSelection(float *buffer, int* index, int size, float value)
{
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x; // Compute flattened global thread ID

    if (threadId < size) // Check if threadId is within bounds
    {
        buffer[threadId] = (threadId == index[0]) ? value : 0; // Set value or zero based on condition
    }
}