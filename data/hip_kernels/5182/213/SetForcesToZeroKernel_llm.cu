#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SetForcesToZeroKernel( float *force, int maxCells )
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation

    int stride = blockDim.x * gridDim.x; // Calculate stride based on grid and block size

    // Loop through the elements with a stride approach for larger data sets
    for(int i = threadId; i < maxCells * 3; i += stride) 
    {
        force[i] = 0.00f;
    }
}