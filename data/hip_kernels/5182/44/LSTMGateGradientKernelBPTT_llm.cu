#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LSTMGateGradientKernelBPTT( float *input, float *previousOutput, float *cellStates,  float *inputGateDeltas, float *forgetGateDeltas, float *outputGateDeltas,  float* outputGateWeightGradient, float* inputGateWeightGradient, float* forgetGateWeightGradient,  int inputCount, int previousOutputCount, int cellsPerBlock )
{
    int weightId = blockDim.x * blockIdx.y * gridDim.x  //rows preceding current row in grid
                  + blockDim.x * blockIdx.x              //blocks preceding current block
                  + threadIdx.x;
    
    int weightsPerGate = inputCount + previousOutputCount + cellsPerBlock + 1;

    if (weightId < weightsPerGate * previousOutputCount / cellsPerBlock)
    {
        int fromId = weightId % weightsPerGate;
        int toId = weightId / weightsPerGate;

        // Using boolean instead of int for flags for better clarity
        bool isFromInputUnit = fromId < inputCount;
        bool isFromPreviousOutputUnit = (fromId >= inputCount) && (fromId < inputCount + previousOutputCount);
        bool isPeephole = (fromId >= inputCount + previousOutputCount) && (fromId < inputCount + previousOutputCount + cellsPerBlock);
        bool isFromBiasUnit = fromId == (inputCount + previousOutputCount + cellsPerBlock);

        // Precompute index offsets to avoid recalculating
        int prevOutputOffset = fromId - inputCount;
        int peepholeOffset = toId * cellsPerBlock + (fromId - inputCount - previousOutputCount);

        // Calculate inputFromWeight with conditional operators
        float inputFromWeight = 
            isFromInputUnit ? input[fromId] :
            isFromPreviousOutputUnit ? previousOutput[prevOutputOffset] :
            isPeephole ? cellStates[peepholeOffset] :
            isFromBiasUnit ? 1.0f : 0.0f;

        // Perform weight gradient computation
        outputGateWeightGradient[weightId] = outputGateDeltas[toId] * inputFromWeight;
        inputGateWeightGradient[weightId] = inputGateDeltas[toId] * inputFromWeight;
        forgetGateWeightGradient[weightId] = forgetGateDeltas[toId] * inputFromWeight;
    }
}