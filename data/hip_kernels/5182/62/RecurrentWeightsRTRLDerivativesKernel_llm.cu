#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RecurrentWeightsRTRLDerivativesKernel(float *previousHiddenActivations, float *hiddenActivationDerivatives, float *recurrentWeights, float *recurrentWeightRTRLDerivatives, float *previousRecurrentWeightRTRLDerivatives)
{
    int partialId = blockDim.x * blockIdx.y * gridDim.x // Calculate unique thread ID
                   + blockDim.x * blockIdx.x
                   + threadIdx.x;

    if (partialId < D_HIDDEN_UNITS * D_HIDDEN_UNITS * D_HIDDEN_UNITS)
    {
        int unitId = partialId / (D_HIDDEN_UNITS * D_HIDDEN_UNITS);
        int weightId = partialId % (D_HIDDEN_UNITS * D_HIDDEN_UNITS);
        int to = weightId / D_HIDDEN_UNITS;
        int from = weightId % D_HIDDEN_UNITS;

        // Use shared memory for partial sums to improve memory access speed
        extern __shared__ float sharedMem[];
        
        float *sumShared = sharedMem;
        
        float sum = 0.0f;
        for (int i = threadIdx.x; i < D_HIDDEN_UNITS; i += blockDim.x)
        {
            sum += recurrentWeights[unitId * D_HIDDEN_UNITS + i] * previousRecurrentWeightRTRLDerivatives[i * (D_HIDDEN_UNITS * D_HIDDEN_UNITS) + weightId];
        }

        sumShared[threadIdx.x] = sum;
        __syncthreads();

        // Perform reduction on shared memory
        if (threadIdx.x == 0)
        {
            float blockSum = 0.0f;
            for (int i = 0; i < blockDim.x; i++)
            {
                blockSum += sumShared[i];
            }
            recurrentWeightRTRLDerivatives[partialId] = hiddenActivationDerivatives[unitId] * ((unitId == to) * previousHiddenActivations[from] + blockSum);
        }
    }
}