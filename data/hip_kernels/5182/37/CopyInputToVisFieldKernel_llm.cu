#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CopyInputToVisFieldKernel(float *input, float *visField, int inputSize)
{
    // Compute global thread ID
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within bounds
    if(threadId < inputSize)
    {
        // Directly copy input to visField
        visField[threadId] = input[threadId];
    }
}