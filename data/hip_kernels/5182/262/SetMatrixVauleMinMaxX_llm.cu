#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for setting matrix values with given conditions
__global__ void SetMatrixVauleMinMaxX(float* matrix, int cols, int size, int id_min, int id_max, float value) {
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Calculate column index
    int id_column = id % cols;
    
    // Set value only if within the specified column range and id is valid
    if (id < size && id_column >= id_min && id_column <= id_max) {
        matrix[id] = value;
    }
}