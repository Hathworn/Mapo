#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InvertPermutationKernel(float* input, float* output, int size)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x; // Optimize block index calculation

    if (id < size) // Check within range early
    {
        int temp = __float2int_rn(input[id]); 

        if (input == output) 
            __syncthreads();

        output[temp] = id;
    }
}