#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixCopy(float* in, float* out, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified id calculation by using 1D grid and block index
    if (id < size)
    {
        out[id] = in[id];
    }
}