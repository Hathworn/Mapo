#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LSTMCellInputGradientKernelBPTT(float *input, float *previousOutput, float *cellInputDeltas, float *cellInputWeightGradient, int inputCount, int previousOutputCount, int cellsPerBlock) 
{
    int weightId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x; // Simplified weightId computation
    
    int weightsPerCell = inputCount + previousOutputCount + 1; // Per cell weight count

    if (weightId < weightsPerCell * previousOutputCount) 
    {
        int fromId = weightId % weightsPerCell;
        int toId = weightId / weightsPerCell;

        float inputFromWeight = 0.0f; // Initialize inputFromWeight
        
        // Conditional assignment
        if (fromId < inputCount) 
        {
            inputFromWeight = input[fromId];
        } 
        else if (fromId < inputCount + previousOutputCount) 
        {
            inputFromWeight = previousOutput[fromId - inputCount];
        } 
        else 
        {
            inputFromWeight = 1.0f; // Bias unit
        }
        
        // Direct calculation without branching
        cellInputWeightGradient[weightId] = cellInputDeltas[toId] * inputFromWeight;
    }
}