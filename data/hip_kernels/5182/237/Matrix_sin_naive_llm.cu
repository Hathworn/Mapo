#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_sin_naive(const float * A, int Acount, int Acols, float * out0, int out0count)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Ensure thread operates only within the bounds
    if (id < out0count) {
        // Use a more efficient math library function if available
        out0[id] = __sinf(A[id]);
    }
}