#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CeilKernel_optimized(const float *A, int Acount, int Acols, float *out0, int out0count)
{
    // Combined calculation for ID to improve performance.
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < out0count)
    {
        out0[id] = ceilf(A[id]); // Use ceilf for better performance with float.
    }
}