#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findMaxIndMultipleDetector(float *input, int* maxInd, int size)
{
    int maxIndex = 0;
    int count = 1;
    float maxValue = input[0];

    // Unroll loop for better performance and use shared memory for partial results
    extern __shared__ int sharedMem[];
    sharedMem[threadIdx.x] = 0;
    __syncthreads();

    for (int i = 1; i < size; i++){
        if (maxValue < input[i]){
            maxIndex = i;
            maxValue = input[i];
            count = 1;
        }
        else if (maxValue == input[i]){
            count++;
        }
    }

    // Write the maxIndex result to shared memory
    sharedMem[threadIdx.x] = (count > 1) ? -1 : maxIndex;
    __syncthreads();

    // Reduce the results in shared memory to get final result
    if (threadIdx.x == 0) {
        maxInd[0] = sharedMem[0];
        for (int i = 1; i < blockDim.x; i++) {
            if (sharedMem[i] != -1) {
                maxInd[0] = sharedMem[i];
                break;
            }
        }
    }
}