#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PlotObserverScaleDownScaleKernel(float* history, int nbCurves, int size)
{
    // Use blockIdx.x with 2D grid configuration and calculate index more efficiently
    int id = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x;

    if (id >= size) return;

    // Pre-calculate base address and optimize memory access
    int baseAddress1 = id;
    int baseAddress2 = baseAddress1 + nbCurves;
    
    // Simplify average calculation
    history[baseAddress1] = (history[baseAddress1] + history[baseAddress2]) * 0.5f;
}