#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Prepare_1_MeansForJoin(float* input, int c_src1, int c_src2, int c_n, float* delta, int imageWidth, int imageHeight)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation

    int size = imageWidth * imageHeight;

    if (id >= size) return; // Early exit if id is out of bounds

    int px = id % imageWidth;
    int py = id / imageWidth;

    bool insideSrc1 = delta[c_src1 * NUM_SUMS * size + 4 * size + id] != 0;
    bool insideSrc2 = delta[c_src2 * NUM_SUMS * size + 4 * size + id] != 0;

    if (input[id] > 0 && (insideSrc1 || insideSrc2)) {

        float2 pixPos = { 2.0f * px / imageWidth - 1, 2.0f * py / imageHeight - 1 };

        // Cache the common index calculation
        int baseIndex = c_n * NUM_SUMS * size + id; 

        // Update delta array with weighted positions and positions squared
        delta[baseIndex + 0 * size] = input[id] * pixPos.x;
        delta[baseIndex + 1 * size] = input[id] * pixPos.y;
        delta[baseIndex + 2 * size] = input[id] * pixPos.x * pixPos.x;
        delta[baseIndex + 3 * size] = input[id] * pixPos.y * pixPos.y;
        delta[baseIndex + 4 * size] = input[id];
    }
    else
    {
        // Cache the common index calculation
        int baseIndex = c_n * NUM_SUMS * size + id;

        // Set all required delta array entries to zero
        delta[baseIndex + 0 * size] = 0;
        delta[baseIndex + 1 * size] = 0;
        delta[baseIndex + 2 * size] = 0;
        delta[baseIndex + 3 * size] = 0;
        delta[baseIndex + 4 * size] = 0;
    }
}