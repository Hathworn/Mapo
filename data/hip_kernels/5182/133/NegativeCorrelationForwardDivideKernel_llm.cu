#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NegativeCorrelationForwardDivideKernel(float* outputPtr, int thisLayerSize, int inputModelCount)
{
    // Calculate global thread ID with optimized grid-stride loop
    int j = blockIdx.x * blockDim.x + threadIdx.x; 
    int stride = blockDim.x * gridDim.x; // Total number of threads

    for (; j < thisLayerSize; j += stride)
    {
        outputPtr[j] /= (float)inputModelCount; // Perform division for each element
    }
}