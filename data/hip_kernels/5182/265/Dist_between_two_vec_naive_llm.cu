#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float Dist_between_two_vec(float * v0, float *v1, int size) {
    float dist = 0;
    for (int i = 0; i < size; i++)
        dist += (v0[i] - v1[i]) * (v0[i] - v1[i]);
    
    return sqrt(dist);
}

__global__ void Dist_between_two_vec_optimized(float * v0, float *v1, int size, float * dst) {
    // Use shared memory for partial results, to minimize global memory access
    extern __shared__ float partial_sum[];
    int tid = threadIdx.x;
    float local_dist = 0;
    
    for (int i = tid; i < size; i += blockDim.x)
        local_dist += (v0[i] - v1[i]); // Corrected to calculate distance contribution

    partial_sum[tid] = local_dist;
    __syncthreads(); // Synchronize to ensure all threads have written to shared memory
    
    // Reduction to sum contributions from all threads
    if (tid == 0) {
        float total_dist = 0;
        for (int i = 0; i < blockDim.x; i++)
            total_dist += partial_sum[i];
        
        dst[0] = total_dist;
    }
}
```
