#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//#define _SIZE_T_DEFINED

extern "C"
{
}

__global__ void ShuffleRGB(float* input, float* output, int size)
{
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    if (id < size)
    {
        // Calculate index using efficient integer math
        int sector = size / 3;
        int index = (id / 3) + (id % 3) * sector;
        output[index] = input[id];
    }
}