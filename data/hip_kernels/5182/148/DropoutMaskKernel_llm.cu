#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DropoutMaskKernel(float *dropoutMaskPtr, float dropout, int inputSize)
{
    // Calculate global thread index
    int i = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    // Ensure the thread index does not exceed input size
    if (i < inputSize)
    {
        // Use ternary operator to simplify condition
        dropoutMaskPtr[i] = (dropout > dropoutMaskPtr[i]) ? 1.0f : 0.0f;
    }
}