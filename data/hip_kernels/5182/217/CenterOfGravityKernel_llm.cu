#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CenterOfGravityKernel(float *pointsCoordinates, float *centerOfGravity, int *activityFlag, int maxCells) 
{
    int threadId = blockDim.x * blockIdx.y * gridDim.x
                 + blockDim.x * blockIdx.x
                 + threadIdx.x;

    // Optimization: Use shared memory for acceleration
    __shared__ float xSumShared;
    __shared__ float ySumShared;
    __shared__ float zSumShared;
    __shared__ int livingCellsShared;

    if (threadId == 0) {
        xSumShared = 0.00f;
        ySumShared = 0.00f;
        zSumShared = 0.00f;
        livingCellsShared = 0;
    }
    __syncthreads();

    float xSum = 0.00f, ySum = 0.00f, zSum = 0.00f;
    int livingCells = 0;
    
    for (int c = threadId; c < maxCells; c += blockDim.x * gridDim.x) {
        if (activityFlag[c] == 1) {
            xSum += pointsCoordinates[c * 3];
            ySum += pointsCoordinates[c * 3 + 1];
            zSum += pointsCoordinates[c * 3 + 2];
            livingCells++;
        }
    }
    
    // Atomic operations to avoid race conditions
    atomicAdd(&xSumShared, xSum);
    atomicAdd(&ySumShared, ySum);
    atomicAdd(&zSumShared, zSum);
    atomicAdd(&livingCellsShared, livingCells);

    __syncthreads();

    if (threadId == 0) {
        centerOfGravity[0] = xSumShared / (float)livingCellsShared;
        centerOfGravity[1] = ySumShared / (float)livingCellsShared;
        centerOfGravity[2] = zSumShared / (float)livingCellsShared;
    }
}