#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ChangeOutputWeightsKernel( float *outputWeights, float *outputWeightDeltas, float *outputDeltas, float *hiddenActivations,  float trainingRate, float momentum )
{
// Using cooperative indexing for grid and block calculations
int weightId = blockIdx.x * blockDim.x + threadIdx.x;

if (weightId < D_OUTPUT_UNITS * D_HIDDEN_UNITS)
{
    // Reduce redundant calculations by storing in register variables
    int to = weightId / D_HIDDEN_UNITS;
    int from = weightId % D_HIDDEN_UNITS;
    float gradient = outputDeltas[to] * hiddenActivations[from];
    float weightDelta = trainingRate * gradient + momentum * outputWeightDeltas[weightId];
    
    // Update global memory efficiently
    outputWeightDeltas[weightId] = weightDelta;
    outputWeights[weightId] += weightDelta;
}
}