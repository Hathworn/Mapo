#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_getRow_FloatPointer_naive(const float * A, int Acount, int Acols, const float * rowId, int empty_par1, int empty_par2, float * out0, int out0count, int out0cols)
{
    // Calculate unique thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread is within matrix column bounds
    if (id < Acols)
    {
        // Directly access the specified row and col
        out0[id] = A[id + static_cast<int>(rowId[0]) * Acols];
    }
}