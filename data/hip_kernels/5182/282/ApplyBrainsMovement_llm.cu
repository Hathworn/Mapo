#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ApplyBrainsMovement(float *CCXY, int dim_XY, float *movement, int dim_movement, int max_clusters) {
    // Compute global thread id
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Check if thread id is within bounds
    if (id < max_clusters) {
        int idx = id * dim_XY;
        
        // Apply movement in XY direction
        if (dim_movement >= 2) {
            CCXY[idx]     -= movement[0];
            CCXY[idx + 1] -= movement[1];
        }

        // Placeholder for rotation in X direction if needed in future
        if (dim_movement >= 3) {
            // Currently no operation
        }
    }
}