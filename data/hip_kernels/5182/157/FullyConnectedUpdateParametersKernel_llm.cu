#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FullyConnectedUpdateParametersKernel(
    float *weightPtr, float *biasPtr, 
    float *weightLearningRatePtr, float *biasLearningRatePtr, 
    float *weightGradientPtr, float *biasGradientPtr, 
    float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize)
{
    // Global thread index in the j-dimension
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread operates only within valid range
    if (j < thisLayerSize)
    {
        // Check dropout mask for the current neuron
        if (!dropoutMaskPtr[j])
        {
            // Use separate variable to reduce conditional complexity
            float *wPtr = weightPtr + j;
            float *wlrPtr = weightLearningRatePtr + j;
            float *wgPtr = weightGradientPtr + j;

            // Vectorialize the weight update loop
            for (int i = 0; i < prevLayerSize; ++i)
            {
                *wPtr -= *wlrPtr * *wgPtr;
                wPtr += thisLayerSize;
                wlrPtr += thisLayerSize;
                wgPtr += thisLayerSize;
            }

            // Update bias for the current neuron
            biasPtr[j] -= biasLearningRatePtr[j] * biasGradientPtr[j];
        }
    }
}