#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Matrix_MultiplElementWise_optimized(const float * A, int Acount, int Acols, const float * B, int Bcount, int Bcols, float * out0, int out0count, int out0cols)
{
    int id = blockDim.x * blockIdx.y * gridDim.x + blockDim.x * blockIdx.x + threadIdx.x;
    if (id < Acount) 
    {
        // Perform element-wise multiplication based on Bcols
        int id_row_col = (Bcols == 1) ? id / Acols : id % Acols;
        out0[id] = A[id] * B[id_row_col];
    }
}