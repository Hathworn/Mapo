#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DrawRgbaTextureKernel(float *target, int targetWidth, int targetHeight, int inputX, int inputY, float *texture, int textureWidth, int textureHeight)
{
    // Calculate the unique identifier for each thread
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    // Precompute number of pixels
    int targetPixels = targetWidth * targetHeight;
    int texturePixels = textureWidth * textureHeight;

    // Calculate the texture channel and pixel position
    int idTextureRgb = id / texturePixels;
    int idTexturePixel = id % texturePixels; // Use modulus instead for clarity
    int idTextureY = idTexturePixel / textureWidth;
    int idTextureX = idTexturePixel % textureWidth; // Likewise use modulus here 

    if (idTextureRgb < 3) // Only process R, G, B channels
    {
        // Swap channels from BGR to RGB
        if (idTextureRgb == 0) idTextureRgb = 2; // Swap R <-> B
        else if (idTextureRgb == 2) idTextureRgb = 0;

        // Continue only if the calculated position is within the target bounds
        if (idTextureX + inputX < targetWidth &&
            idTextureX + inputX >= 0 &&
            idTextureY + inputY < targetHeight &&
            idTextureY + inputY >= 0)
        {
            int tIndex = targetPixels * idTextureRgb + targetWidth * (idTextureY + inputY) + (idTextureX + inputX);
            int aIndex = idTexturePixel + 3 * texturePixels; // Calculate alpha component index

            float a = texture[aIndex];
            target[tIndex] = target[tIndex] * (1.0f - a) + a * texture[id];
        }
    }
}