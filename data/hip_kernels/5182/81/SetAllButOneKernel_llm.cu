#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SetAllButOneKernel(float *buffer, float value, int index, int count)
{
    int threadId = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x; // Calculate global thread ID

    // Check if threadId is within bounds and not equal to index
    if (threadId < count && threadId != index) {
        buffer[threadId] = value; // Assign value to buffer
    }
}