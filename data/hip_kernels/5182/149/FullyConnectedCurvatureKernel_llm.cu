#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FullyConnectedCurvatureKernel(float *weightsGradPtr, float *biasGradPtr, float *shiftedWeightsPtr, float *shiftedBiasPtr, float *avgWeightGradPtr, float *avgBiasGradPtr, float *weightGradCurvePtr, float *biasGradCurvePtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize) {
    // Use local variable to cache values for better performance
    int j = blockDim.x * blockIdx.y * gridDim.x + blockDim.x * blockIdx.x + threadIdx.x;
    
    if (j < thisLayerSize && !dropoutMaskPtr[j]) {
        int index = j;
        float abs_val, avgGrad;
        
        for (int i = 0; i < prevLayerSize; i++) {
            // Precompute avgWeightGradPtr for reduced memory access
            avgGrad = avgWeightGradPtr[index];
            avgGrad = (avgGrad == 0) ? 0.000001 : avgGrad; // Avoid division by zero
            abs_val = abs(weightsGradPtr[index] - shiftedWeightsPtr[index]);
            weightGradCurvePtr[index] = abs_val / avgGrad;
            index += thisLayerSize;
        }

        // Compute bias curvature
        avgGrad = avgBiasGradPtr[j];
        avgGrad = (avgGrad == 0) ? 0.000001 : avgGrad; // Avoid division by zero
        abs_val = abs(biasGradPtr[j] - shiftedBiasPtr[j]);
        biasGradCurvePtr[j] = abs_val / avgGrad;
    }
}