#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define INF 99999.0f  // Define infinity

__device__ float Dist_between_two_vec(float *v0, float *v1, int size) {
    float dist = 0.0f;
    for (int i = 0; i < size; i++)
        dist += (v0[i] - v1[i]) * (v0[i] - v1[i]);
    return sqrtf(dist);
}

__global__ void NN_naive(float *A, int colsA, int sizeA, float *B, int colsB, int numsB, int dim, float *idx, float *dist) {
    float tmp_dist = INF;  // Initialize with defined INF
    int nn_id = -1;
    int idA = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;

    if (idA < sizeA) {  // Boundary check
        for (int idB = 0; idB < numsB; idB++) {
            float adist = Dist_between_two_vec(A + idA * colsA, B + idB * colsB, dim);
            if (tmp_dist > adist) {
                tmp_dist = adist;
                nn_id = idB;  // Keep idB instead of idB * colsB to match data structure
            }
        }
        dist[idA] = tmp_dist;
        idx[idA] = nn_id;
    }
}