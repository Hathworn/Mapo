#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void implantCoeffs(float* matrices, float *coeffArray, int savedCoeffs, int dimsize) {
    int id = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
           + threadIdx.z * blockDim.y * blockDim.x 
           + threadIdx.y * blockDim.x 
           + threadIdx.x;

    int offsetMatrix = id * dimsize * dimsize;
    int offsetCoeff = id * savedCoeffs;
    int coeffsLeft = savedCoeffs;
    int x = 0, y = 0, y_n = 0, x_n = 1;
    int tmp;

    // Directly set the first element
    matrices[offsetMatrix] = coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)];
    coeffsLeft--;

    while (coeffsLeft > 0) {
        x = x_n;
        y = y_n;

        int numberinrow = (x_n < dimsize - 1) ? x_n + 1 : x_n - (y_n - 1);

        while (numberinrow > 0 && coeffsLeft > 0) {
            // Efficient placement
            matrices[offsetMatrix + x + y * dimsize] = coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)];
            numberinrow--;
            coeffsLeft--;

            // Simplified coordinate transformation
            tmp = x;
            x = y;
            y = tmp;

            if ((numberinrow + 1) % 2 == 0) {
                x--;
                y++;
            }
        }

        if (numberinrow % 2 == 1 && coeffsLeft > 0) {
            // Handle extra element for odd row sizes
            matrices[offsetMatrix + x + y * dimsize] = coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)];
            coeffsLeft--;
        }

        // Update coordinate for the next iteration
        if (x_n == dimsize - 1) {
            y_n++;
        } else {
            x_n++;
        }
    }
}