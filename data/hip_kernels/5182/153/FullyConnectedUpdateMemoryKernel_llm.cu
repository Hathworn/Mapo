#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FullyConnectedUpdateMemoryKernel( float *avgWeightGradPtr, float *avgBiasGradPtr, float *avgWeightGradVarPtr, float *avgBiasGradVarPtr, float *weightMemorySizePtr, float *biasMemorySizePtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize )
{
    // Calculate the neuron index in the current layer
    int j = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    
    if (j < thisLayerSize && !dropoutMaskPtr[j]) // Dropout check and bounds check combined
    {
        int index = j;
        float avgWeightGrad, avgWeightGradVar, currentMemSize, updatedMemSize;

        for (int i = 0; i < prevLayerSize; i++)
        {
            // Load values and compute outside the memory access
            avgWeightGrad = avgWeightGradPtr[index];
            avgWeightGradVar = avgWeightGradVarPtr[index];
            currentMemSize = weightMemorySizePtr[index];

            // Update weight memory size
            updatedMemSize = (1.0f - avgWeightGrad * avgWeightGrad / avgWeightGradVar) * currentMemSize + 1.0f;
            weightMemorySizePtr[index] = updatedMemSize;

            index += thisLayerSize;
        }

        // Load values and compute outside the memory access for bias update
        float avgBiasGrad = avgBiasGradPtr[j];
        float avgBiasGradVar = avgBiasGradVarPtr[j];
        float currentBiasMemSize = biasMemorySizePtr[j];

        // Update bias memory size
        biasMemorySizePtr[j] = (1.0f - avgBiasGrad * avgBiasGrad / avgBiasGradVar) * currentBiasMemSize + 1.0f;
    }
}