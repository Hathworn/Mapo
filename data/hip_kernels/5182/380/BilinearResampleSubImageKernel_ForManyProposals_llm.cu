#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BilinearResampleSubImageKernel_ForManyProposals(const float *input, float *output, const float* subImageDefs, bool safeBounds, int subImageDefsDim, int inputWidth, int inputHeight, int outputWidth, int outputHeight, int numberSubImages, int outputSize)
{
    int id = blockDim.x * blockIdx.y * gridDim.x + blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= outputSize) return; // Early exit for out-of-bound threads

    int px = id % outputWidth;  // line in the single output image
    int py = (id / outputWidth) % outputHeight;  // column in the single output image
    int subim_id = id / (outputWidth * outputHeight);  // which image it is

    float subImgCX = subImageDefs[0 + subim_id * subImageDefsDim]; // <-1, 1>
    float subImgCY = subImageDefs[1 + subim_id * subImageDefsDim]; // <-1, 1>
    float subImgDiameter = subImageDefs[2 + subim_id * subImageDefsDim]; // <0,1>

    int maxDiameter = min(inputWidth - 1, inputHeight - 1);
    int diameterPix = (int)(subImgDiameter * maxDiameter);
    diameterPix = max(1, min(maxDiameter, diameterPix)); // Clamp diameterPix within valid bounds

    int subImgX = (int)(inputWidth * (subImgCX + 1) * 0.5f) - diameterPix / 2;
    int subImgY = (int)(inputHeight * (subImgCY + 1) * 0.5f) - diameterPix / 2;

    if (safeBounds)
    {
        subImgX = max(1, min(subImgX, inputWidth - diameterPix - 1));
        subImgY = max(1, min(subImgY, inputHeight - diameterPix - 1));
    }

    float xRatio = (diameterPix - 1) / (float)(outputWidth - 1);
    float yRatio = (diameterPix - 1) / (float)(outputHeight - 1);

    int x = (int)(xRatio * px);
    int y = (int)(yRatio * py);

    if (x + subImgX >= 0 && y + subImgY >= 0 && x + subImgX < inputWidth && y + subImgY < inputHeight)
    {
        //--- X and Y distance difference
        float xDist = (xRatio * px) - x;
        float yDist = (yRatio * py) - y;

        //--- Points
        int baseIndex = (y + subImgY) * inputWidth + x + subImgX;
        float topLeft = input[baseIndex];
        float topRight = input[baseIndex + 1];
        float bottomLeft = input[baseIndex + inputWidth];
        float bottomRight = input[baseIndex + inputWidth + 1];

        float result = topLeft * (1 - xDist) * (1 - yDist) +
                       topRight * xDist * (1 - yDist) +
                       bottomLeft * yDist * (1 - xDist) +
                       bottomRight * xDist * yDist;

        output[py * outputWidth + px + subim_id * outputWidth * outputHeight] = result;
    }
}