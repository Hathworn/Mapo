#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void EstimateParForSubsample(float* subImageDefs, bool safeBounds, int inputWidth, int inputHeight, int2 & subImg, int & diameterPix) {
    diameterPix = (int)(fminf((float)inputWidth, (float)inputHeight) * subImageDefs[2]);

    subImg.x = (int)((float)inputWidth * (subImageDefs[0] + 1) * 0.5f);
    subImg.y = (int)((float)inputHeight * (subImageDefs[1] + 1) * 0.5f);

    int maxDiameter = min(inputWidth - 1, inputHeight - 1);

    diameterPix = max(1, diameterPix);
    diameterPix = min(maxDiameter, diameterPix);

    if (safeBounds) {
        subImg.x = max(subImg.x, 1);
        subImg.y = max(subImg.y, 1);
        subImg.x = min(subImg.x, inputWidth - diameterPix - 1);
        subImg.y = min(subImg.y, inputHeight - diameterPix - 1);
    }
}

__global__ void RetinaTransform_FillRetinaAtomic(float *subImageDefs, float *input, int inputWidth, int inputHeight, float *output, int outputDataSize, float *retinaMask, int retinaDataSize, int retinaMaskColHint, float *retinaDataInserted) {
    int id_pxl = blockDim.x * blockIdx.y * gridDim.x + blockDim.x * blockIdx.x + threadIdx.x;

    int2 subImg;
    int diameterPix;
    bool safeBounds = false;

    int x = id_pxl % inputWidth;
    int y = id_pxl / inputWidth;

    // Calculate parameters for subsample
    EstimateParForSubsample(subImageDefs, safeBounds, inputWidth, inputHeight, subImg, diameterPix);

    if (id_pxl < inputWidth * inputHeight) {
        float minDist = FLT_MAX;  // Use predefined constant for maximum float value
        int minIdx = -1;          // Initialize to -1 for safety

        for (int id_retinaPoint = 0; id_retinaPoint < retinaDataSize; id_retinaPoint++) {
            // Calculate mask coordinates based on diameter and offsets
            float x_mask = (retinaMask[id_retinaPoint * retinaMaskColHint] * diameterPix) + subImg.x;
            float y_mask = (retinaMask[id_retinaPoint * retinaMaskColHint + 1] * diameterPix) + subImg.y;

            float dist = (x - x_mask) * (x - x_mask) + (y - y_mask) * (y - y_mask);

            // Update minimum distance and index if current distance is smaller
            if (dist < minDist) {
                minDist = dist;
                minIdx = id_retinaPoint;
            }
        }

        // Perform atomic operations on the identified minimum index
        atomicAdd(output + minIdx, input[id_pxl]);
        atomicAdd(retinaDataInserted + minIdx, 1);
    }
}