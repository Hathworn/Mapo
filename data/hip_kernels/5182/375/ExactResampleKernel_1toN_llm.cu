#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ExactResampleKernel_1toN(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
    // Calculate the global thread index
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    int size = outputWidth * outputHeight;

    if (id < size)
    {
        // Output point coordinates
        int px = id % outputWidth;
        int py = id / outputWidth;

        // Precompute ratios outside the loop
        float xRatio = static_cast<float>(inputWidth) / outputWidth;
        float yRatio = static_cast<float>(inputHeight) / outputHeight;

        // Corresponding coordinates in the original image (using float operations)
        int x = static_cast<int>(px * xRatio);
        int y = static_cast<int>(py * yRatio);

        output[py * outputWidth + px] = input[y * inputWidth + x];
    }
}