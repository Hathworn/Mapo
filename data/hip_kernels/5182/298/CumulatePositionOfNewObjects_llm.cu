#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CumulatePositionOfNewObjects(float* mask, float* maskNewIds, float* maskOut, int mask_size, int mask_cols, float* centers, int centers_size, int centers_columns) {
    int idx = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;  // Simplified index calculation
    int icol = idx % mask_cols;
    int irow = idx / mask_cols;

    if (idx < mask_size) {
        int i_mask = __float2int_rd(mask[idx]);  // Use fast casting
        int i_obj = __float2int_rd(maskNewIds[i_mask]);  // Use fast casting
        maskOut[idx] = (float)i_obj;  // Cast to float for maskOut

        if (i_obj * centers_columns + 2 < centers_size) {
            atomicAdd(&centers[i_obj * centers_columns + 0], (float)icol);  // Directly reference in atomicAdd
            atomicAdd(&centers[i_obj * centers_columns + 1], (float)irow);
            atomicAdd(&centers[i_obj * centers_columns + 2], 1.0f);
        }
    }
}