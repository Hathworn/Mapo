#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DecreaseErrorAndUtilityKernel(float *localError, float *utility, int *activityFlag, int maxCells, float beta)
{
    // Calculate global threadId
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    // Ensure threadId is within valid range
    if(threadId < maxCells)
    {
        if(activityFlag[threadId] == 1)
        {
            // Update localError and utility using fused multiply-add for efficiency
            localError[threadId] = __fmaf_rn(-beta, localError[threadId], localError[threadId]);
            utility[threadId] = __fmaf_rn(-beta, utility[threadId], utility[threadId]);
        }
    }
}