#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ChangeInputWeightsKernel(float *inputWeights, float *inputWeightDeltas, float *outputWeights, float *outputDeltas, float *inputWeightRTRLDerivatives, float trainingRate, float momentum)
{
    int weightId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    
    if (weightId < D_HIDDEN_UNITS * D_INPUT_UNITS)
    {
        float gradient = 0.0f;

        // Using shared memory for outputWeights part of calculation to reduce global memory access
        __shared__ float sharedOutputWeights[D_HIDDEN_UNITS];
        
        for (int i = 0; i < D_OUTPUT_UNITS; i++)
        {
            float sum = 0.0f;
            
            for (int j = 0; j < D_HIDDEN_UNITS; j++)
            {
                if (threadIdx.x < D_HIDDEN_UNITS)
                {
                    sharedOutputWeights[threadIdx.x] = outputWeights[i * D_HIDDEN_UNITS + j];
                }
                __syncthreads(); // Synchronize threads within block

                sum += sharedOutputWeights[j] * inputWeightRTRLDerivatives[j * D_HIDDEN_UNITS * D_INPUT_UNITS + weightId];
            }
            gradient += outputDeltas[i] * sum;
        }

        float weightDelta = trainingRate * gradient + momentum * inputWeightDeltas[weightId];
        inputWeightDeltas[weightId] = weightDelta;
        inputWeights[weightId] += weightDelta;
    }
}