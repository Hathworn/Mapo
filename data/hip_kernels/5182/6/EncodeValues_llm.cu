#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void EncodeValuesInternal(float value, float& origin, float& dir, float& output, int squaredMode) {
    if (squaredMode == 1) {
        // Compute output using squared mode
        float absValue = fabs(value);
        output = (1 - absValue) * (1 - absValue) * origin + (-value * value + 2 * absValue) * dir;
    } else {
        // Compute output using linear mode
        float absValue = fabs(value);
        output = (1 - absValue) * origin + absValue * dir;
    }
}

__global__ void EncodeValues(float* values, int numOfValues, float* output, int symbolSize, int squaredMode, float* dirX, float* dirY, float* negDirX, float* negDirY, float* originX, float* originY) {
    int threadId = blockDim.x * blockIdx.y * gridDim.x // Rows preceeding current row in grid
                   + blockDim.x * blockIdx.x          // Blocks preceeding current block
                   + threadIdx.x;

    __shared__ float s_values[2];
    
    // Load values into shared memory with clamping if squaredMode is enabled
    if (threadIdx.x < 2) {
        s_values[threadIdx.x] = squaredMode == 1 ? fmaxf(fminf(values[threadIdx.x], 1), -1) : values[threadIdx.x];
    }

    __syncthreads();

    if (threadId >= symbolSize) return;

    // Process X dimension
    float* dir = (s_values[0] > 0) ? dirX : negDirX;
    EncodeValuesInternal(s_values[0], originX[threadId], dir[threadId], output[threadId], squaredMode);

    // Process Y dimension if applicable
    if (numOfValues > 1) {
        dir = (s_values[1] > 0) ? dirY : negDirY;
        EncodeValuesInternal(s_values[1], originY[threadId], dir[threadId], output[threadId], squaredMode);
    }
}