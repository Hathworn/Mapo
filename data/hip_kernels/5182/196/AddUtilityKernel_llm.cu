#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AddUtilityKernel(int s1, int s2, float *distance, float *utility)
{
    // Calculate unique thread index using optimized constants
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    if (threadId < 1)
    {
        // Perform atomic addition to prevent race conditions
        atomicAdd(&utility[s1], distance[s2] - distance[s1]);
    }
}