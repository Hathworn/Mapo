#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DrawRgbaTextureKernel2DBlock(float *target, int targetWidth, int targetHeight, int inputX, int inputY, float *texture, int textureWidth, int textureHeight)
{
    // Using 1D flattened thread ID for better control and coordination
    int id = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * blockIdx.y + gridDim.y * blockIdx.x);

    int targetPixels = targetWidth * targetHeight;

    int texturePixels = textureWidth * textureHeight;

    int idGlobal = id / texturePixels;
    int idTexturePixel = id % texturePixels;

    if (idGlobal < 3) // Process within RGB channels
    {
        int idTextureY = idTexturePixel / textureWidth;
        int idTextureX = idTexturePixel % textureWidth;

        // BGR to RGB channel flip
        int channel = idGlobal == 0 ? 2 : (idGlobal == 2 ? 0 : 1);

        // Ensure texture coordinates are within target bounds
        if (idTextureX + inputX < targetWidth &&
            idTextureX + inputX >= 0 &&
            idTextureY + inputY < targetHeight &&
            idTextureY + inputY >= 0)
        {
            int tIndex = targetPixels * channel + targetWidth * (idTextureY + inputY) + (idTextureX + inputX);
            int aIndex = idTexturePixel + 3 * texturePixels; // A component index
            float a = texture[aIndex];

            // Blending texture color over target color
            target[tIndex] = target[tIndex] * (1.0f - a) + a * texture[id];
        }
    }
}