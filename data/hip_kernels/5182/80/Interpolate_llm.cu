#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Interpolate(float* input1, float* input2, float* output, float weight, int inputSize)
{
    // Optimize thread index calculation using flattening approach
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread executes only if within bounds
    if(threadId < inputSize)
    {
        // Use conditional operator for more concise logic
        output[threadId] = (weight <= 0) ? input1[threadId] :
                           (weight >= 1) ? input2[threadId] :
                           (1 - weight) * input1[threadId] + weight * input2[threadId];
    }
}