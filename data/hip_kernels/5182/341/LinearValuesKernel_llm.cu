#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LinearValuesKernel(const float min, const float max, float* output, const int size, const int shift)
{
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    // Calculate delta once and broadcast to all threads
    float delta = (max - min) / fmaxf((size - 1), 1);
    
    if (id < size)
    {
        output[(id + shift) % size] = min + id * delta;
    }
}