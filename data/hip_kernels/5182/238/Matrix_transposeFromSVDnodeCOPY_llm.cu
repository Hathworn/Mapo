#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_transposeFromSVDnodeCOPY(const float* A, int Acount, int Acols, float* out0)
{
    // Optimize id calculation
    int id = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x;

    int Arows = Acount / Acols;

    if (id < Acount)
    {
        // Calculate x and y only when needed
        int x = id / Arows;
        int y = id % Arows;
        out0[x * Arows + y] = A[y * Acols + x];
    }
}