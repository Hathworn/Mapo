#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BilinearResampleSubImageKernel(float *input, float *output, float* subImageDefs, bool safeBounds, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
    int id = blockDim.x * blockIdx.y * gridDim.x + blockDim.x * blockIdx.x + threadIdx.x;
    int size = outputWidth * outputHeight;

    if (id < size)
    {
        float subImgCX = subImageDefs[0]; // <-1, 1>
        float subImgCY = subImageDefs[1]; // <-1, 1>
        float subImgDiameter = subImageDefs[2]; // <0,1>

        int maxDiameter = min(inputWidth - 1, inputHeight - 1);
        int diameterPix = max(1, min(maxDiameter, (int)(subImgDiameter * maxDiameter)));

        int subImgX = (int)(inputWidth * (subImgCX + 1) * 0.5f) - diameterPix / 2;
        int subImgY = (int)(inputHeight * (subImgCY + 1) * 0.5f) - diameterPix / 2;

        if (safeBounds)
        {
            subImgX = max(subImgX, 1);
            subImgY = max(subImgY, 1);
            subImgX = min(subImgX, inputWidth - diameterPix - 1);
            subImgY = min(subImgY, inputHeight - diameterPix - 1);
        }

        int px = id % outputWidth;
        int py = id / outputWidth;

        float xRatio = (float)(diameterPix - 1) / (outputWidth - 1);
        float yRatio = (float)(diameterPix - 1) / (outputHeight - 1);

        // Calculate x and y only once
        int x = (int)(xRatio * px);
        int y = (int)(yRatio * py);

        if (x + subImgX >= 0 && y + subImgY >= 0 &&
            x + subImgX < inputWidth && y + subImgY < inputHeight)
        {
            // X and Y distance difference
            float xDist = (xRatio * px) - x;
            float yDist = (yRatio * py) - y;

            // Points with boundary checks removed
            int baseIndex = (y + subImgY) * inputWidth + x + subImgX;
            float topLeft = input[baseIndex];
            float topRight = input[baseIndex + 1];
            float bottomLeft = input[baseIndex + inputWidth];
            float bottomRight = input[baseIndex + inputWidth + 1];

            // Bilinear interpolation
            float result = topLeft * (1 - xDist) * (1 - yDist) +
                           topRight * xDist * (1 - yDist) +
                           bottomLeft * yDist * (1 - xDist) +
                           bottomRight * xDist * yDist;

            output[py * outputWidth + px] = result;
        }
    }
}