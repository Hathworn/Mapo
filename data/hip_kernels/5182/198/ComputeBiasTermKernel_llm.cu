#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeBiasTermKernel(float *biasTerm, float cFactor, float *winningFraction, int activeCells, int maxCells)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread ID calculation

    if(threadId < maxCells)
    {
        biasTerm[threadId] = cFactor * (1.00f / activeCells - winningFraction[threadId]);
    }
}