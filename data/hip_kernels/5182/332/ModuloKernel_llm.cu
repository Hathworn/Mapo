#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ModuloKernel(float* input, int divisor, float* output, int size)
{
    // Utilize a more accurate calculation of the global thread ID
    int id = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x;

    // Ensure the id is within the valid range
    if(id < size)
    {
        output[id] = fmodf(input[id], divisor);
    }
}