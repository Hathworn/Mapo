#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MaskInput(float* image, float* mask, float* maskedValues, float* output, int count) {
    // Calculate the unique global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Ensure ID is within the bounds of 'count'
    if (id < count) {
        // Perform computations in one step for efficiency
        output[id] = fmaf(mask[id], image[id] - maskedValues[id], maskedValues[id]);
    }
}