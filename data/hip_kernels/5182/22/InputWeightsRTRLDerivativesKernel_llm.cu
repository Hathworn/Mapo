#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InputWeightsRTRLDerivativesKernel(float *input, float *hiddenActivationDerivatives, float *recurrentWeights, float *inputWeightRTRLDerivatives, float *previousInputWeightRTRLDerivatives)
{
    int partialId = blockDim.x * blockIdx.y * gridDim.x // rows preceding current row in grid
                  + blockDim.x * blockIdx.x             // blocks preceding current block
                  + threadIdx.x;

    // Use shared memory to reduce global memory accesses
    __shared__ float sharedRecurrentWeights[D_HIDDEN_UNITS];
    __shared__ float sharedPreviousRTRLDerivatives[D_HIDDEN_UNITS];

    if (partialId < D_HIDDEN_UNITS * D_HIDDEN_UNITS * D_INPUT_UNITS)
    {
        int unitId = partialId / (D_HIDDEN_UNITS * D_INPUT_UNITS);
        int weightId = partialId % (D_HIDDEN_UNITS * D_INPUT_UNITS);
        int to = weightId / D_INPUT_UNITS;
        int from = weightId % D_INPUT_UNITS;

        float sum = 0;

        // Load data to shared memory
        for (int i = 0; i < D_HIDDEN_UNITS; i++)
        {
            sharedRecurrentWeights[i] = recurrentWeights[unitId * D_HIDDEN_UNITS + i];
            sharedPreviousRTRLDerivatives[i] = previousInputWeightRTRLDerivatives[i * (D_HIDDEN_UNITS * D_INPUT_UNITS) + weightId];
            __syncthreads(); // Synchronize threads to ensure all data is loaded
            
            sum += sharedRecurrentWeights[i] * sharedPreviousRTRLDerivatives[i];
            
            __syncthreads(); // Synchronize threads before next iteration
        }

        inputWeightRTRLDerivatives[partialId] = hiddenActivationDerivatives[unitId] * ((unitId == to) * input[from] + sum);
    }
}