#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Calculate linear function on GPU
__global__ void LinearFunctionKernelDouble(double a1, double a0, double* input, double* output, int size)
{
    // Use 1D block and grid indices for a more direct calculation of the global thread id
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure each thread processes only when within bounds
    if(id < size)
    {
        // Load input value
        double x = input[id];
        
        // Compute and store result
        output[id] = a1 * x + a0;
    }
}