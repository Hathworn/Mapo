#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void IncrementConnectionAgeKernel(int cell, int *connection, int *age, int maxCells)
{
    // Calculate unique thread index
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure threadId is within bounds
    if (threadId < maxCells)
    {
        int index1 = cell * maxCells + threadId;
        int index2 = threadId * maxCells + cell;

        // Check connection and update age
        if (connection[index1] == 1)
        {
            age[index1] += 1;
            age[index2] += 1;
        }
    }
}