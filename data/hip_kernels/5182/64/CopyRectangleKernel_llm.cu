#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CopyRectangleKernel(float *src, int srcOffset, int srcWidth, int srcRectX, int srcRectY, int rectWidth, int rectHeight, float *dest, int destOffset, int destWidth, int destRectX, int destRectY)
{
    // Calculate the 2D thread location in grid
    int localX = blockIdx.x * blockDim.x + threadIdx.x;
    int localY = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within rectangle bounds
    if (localX < rectWidth && localY < rectHeight) {
        // Calculate source and destination positions
        int srcPixelX = srcRectX + localX;
        int srcPixelY = srcRectY + localY;
        int destPixelX = destRectX + localX;
        int destPixelY = destRectY + localY;

        // Perform the copy
        (dest + destOffset)[destPixelX + destPixelY * destWidth] = (src + srcOffset)[srcPixelX + srcPixelY * srcWidth];
    }
}