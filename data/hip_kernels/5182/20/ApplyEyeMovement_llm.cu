#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ApplyEyeMovement(float* currentEye, float* stats, float moveFactor, float scaleFactor, float scaleBase)
{
    float sumWeights = stats[4];

    if (sumWeights > 0)
    {
        // Clamp the movement within bounds [-1, 1]
        currentEye[0] = fmaxf(fminf(moveFactor * stats[0], 1.0f), -1.0f);
        currentEye[1] = fmaxf(fminf(moveFactor * stats[1], 1.0f), -1.0f);

        // Compute variance with optimization for performance
        float variance = sqrtf((stats[2] + stats[3]) * 0.5f);

        // Clamp the scale within bounds [0, 1]
        currentEye[2] = fmaxf(fminf(variance * scaleFactor + scaleBase, 1.0f), 0.0f);
    }
    else
    {
        // Set default eye values when weights are zero
        currentEye[0] = 0.0f;
        currentEye[1] = 0.0f;
        currentEye[2] = 1.0f;
    }
}