#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VectorInputDiffKernel(float *input, int inputSize, float *referenceVector, int maxCells, float *difference)
{
    // Calculate global thread index
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if within bounds
    if (threadId < maxCells * inputSize)
    {
        // Compute the difference
        difference[threadId] = input[threadId % inputSize] - referenceVector[threadId];
    }
}