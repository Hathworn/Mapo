#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Interpolate(float* input1, float* input2, float* output, float weight, int inputSize)
{
    // Calculate threadId more efficiently using grid stride loop
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // total number of threads in the grid

    for (int i = threadId; i < inputSize; i += stride)
    {
        // Efficient conditional computation without additional checks
        output[i] = weight <= 0 ? input1[i] : (weight >= 1 ? input2[i] : (1 - weight) * input1[i] + weight * input2[i]);
    }
}