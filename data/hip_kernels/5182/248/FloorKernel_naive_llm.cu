#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void FloorKernel_optimized(const float * A, int Acount, int Acols, float * out0, int out0count)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D grid instead of 2D
    int stride = blockDim.x * gridDim.x;            // Calculate stride for processing
    for (int i = id; i < out0count; i += stride)    // Loop to process multiple elements per thread
    {
        out0[i] = floor(A[i]);
    }
}