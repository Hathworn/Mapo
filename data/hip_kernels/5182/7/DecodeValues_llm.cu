#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DecodeValues(float* superposition, int symbolSize, float* output, float* reliability, int numOfValues, int squaredMode, float* dirX, float* dirY, float* negDirX, float* negDirY, float* originX, float* originY)
{
    int threadId = blockDim.x * blockIdx.x + threadIdx.x + blockDim.x * gridDim.x * blockIdx.y; // Calculate threadId

    if (threadId >= numOfValues)
        return;

    float* dir = threadId == 0 ? dirX : dirY;
    float* negDir = threadId == 0 ? negDirX : negDirY;
    float* origin = threadId == 0 ? originX : originY;

    float localOutput = 0.0f;
    float localReliability = 0.0f;

    #pragma unroll
    for (int i = 0; i < symbolSize; i++)
    {
        // Use fused multiply-add for better precision and performance
        localOutput = fmaf(superposition[i], dir[i], localOutput);
        localOutput = fmaf(-superposition[i], negDir[i], localOutput);

        localReliability = fmaf(superposition[i], origin[i], localReliability);
    }

    localReliability += fabs(localOutput);
    output[threadId] = localOutput / localReliability;
    reliability[threadId] = localReliability;
}