#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AdaptRefVectorKernel(int cell, float *referenceVector, float oldErrorFraction, float youngErrorFraction, float decayFactor, int *winningCount, float *difference, int inputSize)
{
    // Calculate thread ID in a more efficient way
    int threadId = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    if (threadId < inputSize)
    {
        // Precalculate exponential value to avoid recalculating inside the loop
        float expValue = expf(-decayFactor * winningCount[cell]);
        float errorFraction = (youngErrorFraction - oldErrorFraction) * expValue + oldErrorFraction;
        referenceVector[cell * inputSize + threadId] += errorFraction * difference[cell * inputSize + threadId];
    }
}