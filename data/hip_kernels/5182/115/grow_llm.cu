#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grow(float *matrices, const int dimension, const int coefficients, const int population, float *chromosomes, const float * noise, const float mutationRate, const int kept, const float* fitnesses, int *mark, const int alpha) {

    int i, wloc;
    hiprandState st;

    int threadGlobalID = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
                         + threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;

    int chromOffset = threadGlobalID * coefficients;

    if (threadGlobalID >= population) return;  // Prevent out-of-bound access

    hiprand_init((int)noise[threadGlobalID] << threadGlobalID, threadGlobalID * (threadGlobalID == population - 1 ? noise[0] : noise[threadGlobalID]), 0, &st);

    if (threadGlobalID >= kept) { // Combined repopulate and mutate
        int parent1 = floor(hiprand_uniform(&st) * kept);
        int parent2 = floor(hiprand_uniform(&st) * kept);
        int point = floor(hiprand_uniform(&st) * coefficients);

        for (i = 0; i < coefficients; i++) {
            chromosomes[chromOffset + i] = (i < point ? chromosomes[parent1 * coefficients + i] : chromosomes[parent2 * coefficients + i]);

            if (hiprand_uniform(&st) <= mutationRate) { // Mutate
                chromosomes[chromOffset + i] *= (hiprand_uniform(&st) < 0.5 ? -alpha : alpha);
            }
        }
    }

    if (threadGlobalID < coefficients) {
        for (i = 0; i < population; i++) {
            float probability = 1 - sqrtf((fitnesses[i] - fitnesses[population - 1]) / (fitnesses[0] - fitnesses[population - 1]));
            mark[coefficients * i + threadGlobalID] = hiprand_uniform(&st) < probability ? 1 : 0;
        }

        wloc = -1;
        float tmp1;

        for (i = 0; i < population; i++) {
            if (mark[coefficients * i + threadGlobalID] == 1) {
                if (wloc == -1) {
                    wloc = i;
                    tmp1 = chromosomes[coefficients * i + threadGlobalID];
                } else {
                    float tmp2 = chromosomes[coefficients * i + threadGlobalID];
                    chromosomes[coefficients * i + threadGlobalID] = tmp1;
                    tmp1 = tmp2;
                }
            }
        }
        if (wloc != -1) {
            chromosomes[coefficients * wloc + threadGlobalID] = tmp1;
        }
    }

    __syncthreads();
    
    for (i = 0; i < dimension * dimension; i++) {
        matrices[threadGlobalID * dimension * dimension + i] = 0.0f;
    }
}