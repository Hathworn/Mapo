#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TwoNodesDifferenceKernel(int nodeOne, int nodeTwo, int vectorLength, float *referenceVector, float *twoNodesDifference)
{
    // Use a 1D grid of 1D blocks for simplified thread indexing
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Only compute if threadId is within bounds
    if (threadId < vectorLength)
    {
        // Calculate difference for the corresponding elements in referenceVector
        twoNodesDifference[threadId] = referenceVector[nodeOne * vectorLength + threadId] - referenceVector[nodeTwo * vectorLength + threadId];
    }
}