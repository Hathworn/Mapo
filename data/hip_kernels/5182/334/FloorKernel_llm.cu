#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FloorKernel(float* input, float* output, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified global thread index calculation
    if(id < size)
    {
        output[id] = floorf(input[id]); // Use floorf for single precision float optimization
    }
}