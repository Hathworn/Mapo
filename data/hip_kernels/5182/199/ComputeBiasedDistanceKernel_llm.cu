#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel
__global__ void ComputeBiasedDistanceKernel(float *distance, float *biasedDistance, float *biasTerm, int maxCells)
{
    // Efficiently calculate the global thread index
    int threadId = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x;

    // Avoid divergence with a more efficient check
    if (threadId < maxCells)
    {
        // Direct memory access within the given bounds
        biasedDistance[threadId] = distance[threadId] + biasTerm[threadId];
    }
}