#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GaussianMinMaxField(float* input, int inputCount, float* mins, float* maxes)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation

    if (i < inputCount) // Boundary check
    {
        float value = input[i]; // Load input value to register to avoid multiple reads
        atomicMin(&mins[i], value); // Use atomic operations for min
        atomicMax(&maxes[i], value); // Use atomic operations for max
    }
}