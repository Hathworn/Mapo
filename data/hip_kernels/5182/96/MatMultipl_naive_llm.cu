#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatMultipl_optimized(float *A, float *B, float *C, int nColsA, int nColsB, int sizeC) {
    int i_col = blockIdx.x * blockDim.x + threadIdx.x; // Compute column index
    int i_row = blockIdx.y * blockDim.y + threadIdx.y; // Compute row index

    if (i_col < nColsB && i_row < sizeC / nColsB) { // Bounds check
        float Cvalue = 0.0f;
        for (int e = 0; e < nColsA; e++) {
            Cvalue += A[i_row * nColsA + e] * B[e * nColsB + i_col];
        }
        C[i_row * nColsB + i_col] = Cvalue; // Update C
    }
}