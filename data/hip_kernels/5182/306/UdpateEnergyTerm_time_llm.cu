#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UdpateEnergyTerm_time(float* energy, int energy_dim, int nPatches, float* idFocuser_focused, float par_time_increase_energy_on_focus, float par_time_decrease_energy_in_time) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified global thread ID calculation
    if (id < energy_dim * nPatches) {  // Ensure id is within bounds
        int idDim = id % energy_dim;
        int idPatch = id / energy_dim;

        if (idDim == 0) {  // Check if this is the 'time' dimension
            if (idPatch == static_cast<int>(*idFocuser_focused)) {  // Check for focus
                energy[id] += par_time_increase_energy_on_focus;
            } else {
                energy[id] /= par_time_decrease_energy_in_time;
            }
        }
    }
}