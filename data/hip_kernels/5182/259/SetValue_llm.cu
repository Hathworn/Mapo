#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SetValue(float* im, float val, int size) 
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified ID calculation
    if (id < size) 
    {
        im[id] = val; // Set value without branching
    }
}