#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LSTMDeltaKernelBPTT( float* deltas, float* cellStates, float* previousCellStates, float* cellStateErrors, float* nextCellStateErrors,  float* outputGateDeltas, float* forgetGateDeltas, float* nextForgetGateDeltas, float* inputGateDeltas, float* nextInputGateDeltas, float* cellInputDeltas,  float* cellInputActivations, float* cellStateActivations, float* outputGateActivations, float* nextForgetGateActivations, float* inputGateActivations,  float* cellInputActivationDerivatives, float* cellStateActivationDerivatives, float* outputGateActivationDerivatives, float* forgetGateActivationDerivatives, float* inputGateActivationDerivatives,  float* cellInputWeights, float* outputGateWeights, float* forgetGateWeights, float* inputGateWeights,  int inputCount, int cellCount, int cellsPerBlock )
{
    int memoryBlockId = blockDim.x * blockIdx.y * gridDim.x   //rows preceding current row in grid
                      + blockDim.x * blockIdx.x              //blocks preceding current block
                      + threadIdx.x;

    if (memoryBlockId < cellCount / cellsPerBlock)
    {
        // Initialize outputGateDeltas for memoryBlockId
        outputGateDeltas[memoryBlockId] = 0;
        float tempOutputDelta = 0; // Use a local variable for reduction
        for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
        {
            tempOutputDelta += cellStateActivations[cellId] * deltas[cellId];
        }
        outputGateDeltas[memoryBlockId] = tempOutputDelta * outputGateActivationDerivatives[memoryBlockId];

        // Loop through cells within the block
        for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
        {
            int relativeCellId = cellId - (memoryBlockId * cellsPerBlock);
            int peepHoleWeightId = (memoryBlockId * (inputCount + cellCount + cellsPerBlock + 1)) + inputCount + cellCount + relativeCellId;
            float cellStateError = deltas[cellId] * outputGateActivations[memoryBlockId] * cellStateActivationDerivatives[cellId] +
                                   nextCellStateErrors[cellId] * nextForgetGateActivations[memoryBlockId] +
                                   nextInputGateDeltas[memoryBlockId] * inputGateWeights[peepHoleWeightId] +
                                   nextForgetGateDeltas[memoryBlockId] * forgetGateWeights[peepHoleWeightId] +
                                   outputGateDeltas[memoryBlockId] * outputGateWeights[peepHoleWeightId];
            cellStateErrors[cellId] = cellStateError;

            cellInputDeltas[cellId] = inputGateActivations[memoryBlockId] * cellInputActivationDerivatives[cellId] * cellStateErrors[cellId];
        }

        // Initialize gate deltas
        float tempInputDelta = 0;
        float tempForgetDelta = 0;
        for (int cellId = memoryBlockId * cellsPerBlock; cellId < (memoryBlockId + 1) * cellsPerBlock; cellId++)
        {
            tempInputDelta += cellStateErrors[cellId] * cellInputActivations[cellId];
            tempForgetDelta += cellStateErrors[cellId] * previousCellStates[cellId];
        }
        inputGateDeltas[memoryBlockId] = tempInputDelta * inputGateActivationDerivatives[memoryBlockId];
        forgetGateDeltas[memoryBlockId] = tempForgetDelta * forgetGateActivationDerivatives[memoryBlockId];
    }
}