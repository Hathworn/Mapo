#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float activationProbability(float x, float sigma) {
    return 1.0 / (1.0 + expf(-sigma * x));
}

// Optimized RBMForwardKernel
__global__ void RBMForwardKernel(float *inputPtr, float *outputPtr, float *weightPtr, float *biasPtr, float sigma, int prevLayerSize, int thisLayerSize, bool useDropoutMask, bool useDropout, float dropoutRate, float *dropoutMask) {
    int j = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x; // Calculate the index j

    if (j < thisLayerSize) {
        if (useDropout && useDropoutMask && !dropoutMask[j]) {
            outputPtr[j] = 0; // Set output to 0 if dropout is applied
        } else {
            float sum = biasPtr[j]; // Start sum with bias
            int index = j;

            for (int i = 0; i < prevLayerSize; ++i) {
                sum += weightPtr[index] * inputPtr[i];
                index += thisLayerSize;
            }

            float result = activationProbability(sum, sigma);

            // Adjust result when dropout is used without mask
            if (useDropout && !useDropoutMask && dropoutRate < 1) {
                result /= dropoutRate;
            }

            outputPtr[j] = result; // Assign computed result to output
        }
    }
}