#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ApplyThreshold(float* probabilitiesInputs, float* binaryOutput, float* probability, int count) {
    // Optimize index calculation to use built-in function for better readability
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < count) {
        // Use ternary operator for branchless optimized assignment
        binaryOutput[id] = (probabilitiesInputs[id] < probability[0]) ? 0.0f : 1.0f;
    }
}