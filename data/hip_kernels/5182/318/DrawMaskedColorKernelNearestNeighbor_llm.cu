#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DrawMaskedColorKernelNearestNeighbor(float *target, int targetWidth, int targetHeight, int inputX, int inputY, float *texture, int textureWidth, int textureHeight, int objectWidth, int objectHeight, float r, float g, float b )
{
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    int targetPixels = targetWidth * targetHeight;
    int objectPixels = objectWidth * objectHeight;
    
    int idObjectRgb = id / objectPixels;
    if (idObjectRgb >= 3) return; // Early exit if not within RGB range

    int idObjectPixel = id % objectPixels;
    int idObjectY = idObjectPixel / objectWidth;
    int idObjectX = idObjectPixel % objectWidth;

    int targetRgb = idObjectRgb;
    switch (idObjectRgb) {
        case 0: targetRgb = 2; break; // R to B
        case 2: targetRgb = 0; break; // B to R
    }

    int targetX = idObjectX + inputX;
    int targetY = idObjectY + inputY;
    if (targetX < targetWidth && targetX >= 0 && targetY < targetHeight && targetY >= 0) {
        int textureX = textureWidth * idObjectX / objectWidth;
        int textureY = textureHeight * idObjectY / objectHeight;
        int textureId = textureY * textureWidth + textureX;

        int aIndex = textureId + 3 * textureWidth * textureHeight; // A component index
        float a = texture[aIndex];
        
        if (a > 0) {
            int tIndex = targetPixels * targetRgb + targetWidth * targetY + targetX;
            float color = (idObjectRgb == 0) ? r : ((idObjectRgb == 1) ? g : b); // Select color component
            target[tIndex] = target[tIndex] * (1.0f - a) + a * color;
        }
    }
}