#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeQuadsKernel(float *pointsCoordinates, float *vertexData, int quadOffset, float textureSide, int *activityFlag, int textureWidth, int maxCells) {
    int threadId = blockDim.x * blockIdx.y * gridDim.x + blockDim.x * blockIdx.x + threadIdx.x;
    
    if (threadId < maxCells) {
        float x = pointsCoordinates[threadId * 3];
        float y = pointsCoordinates[threadId * 3 + 1];
        float z = pointsCoordinates[threadId * 3 + 2];
        
        // Precompute reused values to reduce operations within the loop
        float halfSide = (activityFlag[threadId] == 0) ? 0.00f : 0.50f * textureSide;
        int baseQuadOffset = quadOffset + threadId * 36;
        int baseTextureOffset = quadOffset + maxCells * 4 * 3 * 3 + threadId * 24;
        float textureAbsLength = (float)(maxCells * textureWidth);
        float textureCoord = (float)(threadId * textureWidth) / textureAbsLength;
        float textureCoordNext = (float)((threadId + 1) * textureWidth) / textureAbsLength;

        // Vertical x-aligned
        vertexData[baseQuadOffset] = x - halfSide;
        vertexData[baseQuadOffset + 1] = y + halfSide;
        vertexData[baseQuadOffset + 2] = z;
        vertexData[baseTextureOffset] = textureCoord;
        vertexData[baseTextureOffset + 1] = 0.00f;

        vertexData[baseQuadOffset + 3] = x - halfSide;
        vertexData[baseQuadOffset + 4] = y - halfSide;
        vertexData[baseQuadOffset + 5] = z;
        vertexData[baseTextureOffset + 2] = textureCoord;
        vertexData[baseTextureOffset + 3] = 1.00f;

        vertexData[baseQuadOffset + 6] = x + halfSide;
        vertexData[baseQuadOffset + 7] = y - halfSide;
        vertexData[baseQuadOffset + 8] = z;
        vertexData[baseTextureOffset + 4] = textureCoordNext;
        vertexData[baseTextureOffset + 5] = 1.00f;

        vertexData[baseQuadOffset + 9] = x + halfSide;
        vertexData[baseQuadOffset + 10] = y + halfSide;
        vertexData[baseQuadOffset + 11] = z;
        vertexData[baseTextureOffset + 6] = textureCoordNext;
        vertexData[baseTextureOffset + 7] = 0.00f;

        // Horizontal
        vertexData[baseQuadOffset + 12] = x - halfSide;
        vertexData[baseQuadOffset + 13] = y;
        vertexData[baseQuadOffset + 14] = z + halfSide;
        vertexData[baseTextureOffset + 8] = textureCoord;
        vertexData[baseTextureOffset + 9] = 1.00f;

        vertexData[baseQuadOffset + 15] = x - halfSide;
        vertexData[baseQuadOffset + 16] = y;
        vertexData[baseQuadOffset + 17] = z - halfSide;
        vertexData[baseTextureOffset + 10] = textureCoord;
        vertexData[baseTextureOffset + 11] = 0.00f;

        vertexData[baseQuadOffset + 18] = x + halfSide;
        vertexData[baseQuadOffset + 19] = y;
        vertexData[baseQuadOffset + 20] = z - halfSide;
        vertexData[baseTextureOffset + 12] = textureCoordNext;
        vertexData[baseTextureOffset + 13] = 0.00f;

        vertexData[baseQuadOffset + 21] = x + halfSide;
        vertexData[baseQuadOffset + 22] = y;
        vertexData[baseQuadOffset + 23] = z + halfSide;
        vertexData[baseTextureOffset + 14] = textureCoordNext;
        vertexData[baseTextureOffset + 15] = 1.00f;

        // Vertical z-aligned
        vertexData[baseQuadOffset + 24] = x;
        vertexData[baseQuadOffset + 25] = y - halfSide;
        vertexData[baseQuadOffset + 26] = z + halfSide;
        vertexData[baseTextureOffset + 16] = textureCoordNext;
        vertexData[baseTextureOffset + 17] = 1.00f;

        vertexData[baseQuadOffset + 27] = x;
        vertexData[baseQuadOffset + 28] = y - halfSide;
        vertexData[baseQuadOffset + 29] = z - halfSide;
        vertexData[baseTextureOffset + 18] = textureCoord;
        vertexData[baseTextureOffset + 19] = 1.00f;

        vertexData[baseQuadOffset + 30] = x;
        vertexData[baseQuadOffset + 31] = y + halfSide;
        vertexData[baseQuadOffset + 32] = z - halfSide;
        vertexData[baseTextureOffset + 20] = textureCoord;
        vertexData[baseTextureOffset + 21] = 0.00f;

        vertexData[baseQuadOffset + 33] = x;
        vertexData[baseQuadOffset + 34] = y + halfSide;
        vertexData[baseQuadOffset + 35] = z + halfSide;
        vertexData[baseTextureOffset + 22] = textureCoordNext;
        vertexData[baseTextureOffset + 23] = 0.00f;
    }
}