#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Block and thread indexing optimization
__global__ void InitCentroidsKernel(float *centroidCoordinates, float *randomNumbers, float minX, float maxX, float minY, float maxY, int centroids) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure threadId is within bounds
    if (threadId < centroids) {
        centroidCoordinates[threadId * 2] = (maxX - minX) * randomNumbers[threadId * 2] + minX;
        centroidCoordinates[threadId * 2 + 1] = (maxY - minY) * randomNumbers[threadId * 2 + 1] + minY;
    }
}