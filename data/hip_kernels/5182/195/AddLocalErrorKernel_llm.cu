#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AddLocalErrorKernel(int s1, float *distance, float *localError)
{
    // Calculate the unique thread index
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Reduce unnecessary computations by removing gridDim.x and blockIdx.y
    if (threadId == 0)
    {
        // Use atomicAdd to ensure correct concurrent writes
        atomicAdd(&localError[s1], distance[s1] * distance[s1]);
    }
}