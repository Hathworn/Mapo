#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeHistogramKernel(float *globalMemData, int *globalHist) {
    int globalThreadId = blockDim.x * blockIdx.y * gridDim.x  // rows preceding current row in grid
                       + blockDim.x * blockIdx.x               // blocks preceding current block
                       + threadIdx.x;
    int localThreadId = threadIdx.x;
    extern __shared__ int partialHist[];

    // Initialize shared memory only if within D_BINS
    if (localThreadId < D_BINS) {
        partialHist[localThreadId] = 0;
    }
    __syncthreads();

    // Operate only if within bounds
    if (globalThreadId < D_MEMORY_BLOCK_SIZE) {
        float myLocalDataValue = globalMemData[globalThreadId];
        int binIdToWrite = 0 + (D_BINS - 1) * (myLocalDataValue > D_MAX_VALUE);

        // Check if within valid range
        if (myLocalDataValue >= D_MIN_VALUE && myLocalDataValue <= D_MAX_VALUE) {
            // Calculate bin index
            float biasedValue = myLocalDataValue - D_MIN_VALUE;
            binIdToWrite = (int)floor((double)(biasedValue / D_BIN_VALUE_WIDTH)) + 1;

            // Special case for maximum value
            if (myLocalDataValue == D_MAX_VALUE) {
                binIdToWrite = D_BINS - 2;
            }
        }
        // Update local histogram with atomic operation
        atomicAdd(&(partialHist[binIdToWrite]), 1);
        __syncthreads();

        // Copy local histogram to global histogram
        if (localThreadId < D_BINS) {
            atomicAdd(&(globalHist[localThreadId]), partialHist[localThreadId]);
        }
    }
}