#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RBMInputForwardKernel(float *inputPtr, float *outputPtr, float *biasPtr, bool applyBias, int thisLayerSize)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread is within bounds
    if (i < thisLayerSize)
    {
        float result = inputPtr[i];
        
        // Conditionally add the bias
        if (applyBias)
        {
            result += biasPtr[i];
        }
        
        // Store the result in output
        outputPtr[i] = result;  // Correct the output assignment
    }
}