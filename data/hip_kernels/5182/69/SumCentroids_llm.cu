#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SumCentroids(float* delta, float* sumDelta, int numOfCentroids, int numOfElements)
{
    // Use blockDim.x * blockDim.y from grid stride loop to handle multiple blocks in one kernel
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int idx = id; idx < numOfCentroids; idx += stride) 
    {
        float sum = 0;
        // Optimize memory access by contiguous access with stride
        for (int i = 0; i < numOfElements; i++) 
        {
            sum += delta[numOfElements * idx + i];
        }
        sumDelta[idx] = sum;
    }
}