#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ExponentialFunctionKernel(float exponent, float* input, float* output, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation

    if (id < size)
    {
        output[id] = pow(input[id], exponent); // Utilize id directly for efficiency
    }
}