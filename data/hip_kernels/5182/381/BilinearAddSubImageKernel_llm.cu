#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BilinearAddSubImageKernel(float *input, float *opImage, float* subImageDefs, int inputWidth, int inputHeight, int opImageWidth, int opImageHeight)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified ID calculation

    float subImgCX = subImageDefs[0];
    float subImgCY = subImageDefs[1];
    float subImgDiameter = subImageDefs[2];

    int maxDiameter = min(inputWidth, inputHeight);
    int diameterPix = max(1, int(subImgDiameter * maxDiameter));

    int subImgX = int(inputWidth * (subImgCX + 1) * 0.5f) - diameterPix / 2;
    int subImgY = int(inputHeight * (subImgCY + 1) * 0.5f) - diameterPix / 2;

    int px = id % diameterPix;
    int py = id / diameterPix;

    if (px + subImgX >= 0 && py + subImgY >= 0 && px + subImgX < inputWidth && py + subImgY < inputHeight && py < diameterPix)
    {
        float xRatio = float(opImageWidth - 1) / diameterPix;
        float yRatio = float(opImageHeight - 1) / diameterPix;

        int x = int(xRatio * px);
        int y = int(yRatio * py);

        float xDist = (xRatio * px) - x;
        float yDist = (yRatio * py) - y;

        // Access opImage safely with boundary check
        float topLeft = opImage[y * opImageWidth + x];
        float topRight = (x + 1 < opImageWidth) ? opImage[y * opImageWidth + x + 1] : 0.0f;
        float bottomLeft = (y + 1 < opImageHeight) ? opImage[(y + 1) * opImageWidth + x] : 0.0f;
        float bottomRight = (x + 1 < opImageWidth && y + 1 < opImageHeight) ? opImage[(y + 1) * opImageWidth + x + 1] : 0.0f;

        float result = topLeft * (1 - xDist) * (1 - yDist) +
                       topRight * xDist * (1 - yDist) +
                       bottomLeft * yDist * (1 - xDist) +
                       bottomRight * xDist * yDist;

        atomicAdd(&input[(py + subImgY) * inputWidth + px + subImgX], result); // Safely accumulate result
    }
}