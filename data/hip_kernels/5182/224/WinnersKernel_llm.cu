#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void WinnersKernel(float *winner, float *vertexData, int vertexOffset, float *pointsCoordinates, float cubeSize, int maxCells)
{
    int threadId = blockDim.x * blockIdx.y * gridDim.x // Calculate unique thread ID
                   + blockDim.x * blockIdx.x
                   + threadIdx.x;

    if (threadId < maxCells && winner[threadId] == 1.00f)
    {
        // Pre-calculate indices
        int coordIdx = threadId * 3;
        int vertexBaseIdx = vertexOffset;

        // Load point coordinates into registers (faster access)
        float x = pointsCoordinates[coordIdx];
        float y = pointsCoordinates[coordIdx + 1];
        float z = pointsCoordinates[coordIdx + 2];

        // Pre-calculate constants
        float side = 1.2f * cubeSize;
        float halfSize = 0.50f * side;

        // Calculate and store vertex data
        vertexData[vertexBaseIdx] = x - halfSize;
        vertexData[vertexBaseIdx + 1] = y - halfSize;
        vertexData[vertexBaseIdx + 2] = z - halfSize;
        vertexData[vertexBaseIdx + 3] = x - halfSize;
        vertexData[vertexBaseIdx + 4] = y - halfSize;
        vertexData[vertexBaseIdx + 5] = z + halfSize;
        vertexData[vertexBaseIdx + 6] = x + halfSize;
        vertexData[vertexBaseIdx + 7] = y - halfSize;
        vertexData[vertexBaseIdx + 8] = z + halfSize;
        vertexData[vertexBaseIdx + 9] = x + halfSize;
        vertexData[vertexBaseIdx + 10] = y - halfSize;
        vertexData[vertexBaseIdx + 11] = z - halfSize;
    }
}