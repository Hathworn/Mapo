#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Copy_matA_to_matB_withShuffleIdx(float *A, float *B, int size, int cols, float *new_idxs, int max_rows) {
    // Calculate unique thread ID in the grid
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    
    // Calculate row and column indices
    int irow = id / cols;
    int icol = id % cols;

    // Check if within valid range
    if (id < size) {
        // Reverse row index for output
        int irow_new = max_rows - 1 - irow;
        
        // Fetch the old row index from new_idxs
        int irow_old = __float2int_rd(new_idxs[irow]); // Use efficient conversion
        
        // Copy data from A to B with re-mapped indices
        B[irow_new * cols + icol] = A[irow_old * cols + icol];
    }
}