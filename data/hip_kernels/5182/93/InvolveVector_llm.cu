#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InvolveVector(float* input, float* output, int inputSize)
{
    // Calculate a global thread index
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread index is within the valid range
    if(threadId < inputSize)
    {
        output[threadId] = input[inputSize - threadId - 1];
    }
}