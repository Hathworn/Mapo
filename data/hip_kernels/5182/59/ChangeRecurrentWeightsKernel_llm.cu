#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ChangeRecurrentWeightsKernel(float *recurrentWeights, float *recurrentWeightDeltas, float *outputWeights, float *outputDeltas, float *recurrentWeightRTRLDerivatives, float trainingRate, float momentum)
{
    int weightId = blockDim.x * blockIdx.y * gridDim.x // Calculate global weight index
                   + blockDim.x * blockIdx.x
                   + threadIdx.x;

    if (weightId < D_HIDDEN_UNITS * D_HIDDEN_UNITS)
    {
        float gradient = 0;

        // Optimize loop by computing address outside the loop
        int baseHiddenUnits = D_HIDDEN_UNITS * weightId;

        for (int i = 0; i < D_OUTPUT_UNITS; i++)
        {
            float sum = 0;

            // Optimize addressing by using a pointer to recurrentWeightRTRLDerivatives
            float *derivativePtr = recurrentWeightRTRLDerivatives + baseHiddenUnits;

            // Unroll loop to reduce iteration overhead
            for (int j = 0; j < D_HIDDEN_UNITS; j += 2)
            {
                sum += outputWeights[i * D_HIDDEN_UNITS + j] * derivativePtr[j];
                if (j + 1 < D_HIDDEN_UNITS)
                    sum += outputWeights[i * D_HIDDEN_UNITS + j + 1] * derivativePtr[j + 1];
            }

            gradient += outputDeltas[i] * sum;
        }

        float weightDelta = trainingRate * gradient + momentum * recurrentWeightDeltas[weightId];
        recurrentWeightDeltas[weightId] = weightDelta;
        recurrentWeights[weightId] += weightDelta;
    }
}