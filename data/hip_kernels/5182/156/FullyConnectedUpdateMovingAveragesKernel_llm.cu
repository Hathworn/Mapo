#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FullyConnectedUpdateMovingAveragesKernel( float *weightsGradPtr, float *biasGradPtr, float *weightsGradCurvePtr, float *biasGradCurvePtr, float *avgWeightGradPtr, float *avgBiasGradPtr, float *avgWeightGradVarPtr, float *avgBiasGradVarPtr, float *avgWeightGradCurvePtr, float *avgBiasGradCurvePtr, float *avgWeightGradCurveVarPtr, float *avgBiasGradCurveVarPtr, float *weightMemorySizePtr, float *biasMemorySizePtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize )
{
    int j = blockDim.x * blockIdx.y * gridDim.x + blockDim.x * blockIdx.x + threadIdx.x;

    if (j < thisLayerSize && !dropoutMaskPtr[j]) // Early exit if j is out of bounds or mask is active
    {
        float invWeightMemSize, invBiasMemSize;
        int index = j;

        invBiasMemSize = 1.0f / biasMemorySizePtr[j];
        
        for (int i = 0; i < prevLayerSize; i++)
        {
            invWeightMemSize = 1.0f / weightMemorySizePtr[index];

            // Update moving averages for weights
            float weightGrad = weightsGradPtr[index];
            float weightGradCurve = weightsGradCurvePtr[index];
            avgWeightGradPtr[index] = (1.0f - invWeightMemSize) * avgWeightGradPtr[index] + invWeightMemSize * weightGrad;
            avgWeightGradVarPtr[index] = (1.0f - invWeightMemSize) * avgWeightGradVarPtr[index] + invWeightMemSize * weightGrad * weightGrad;
            avgWeightGradCurvePtr[index] = (1.0f - invWeightMemSize) * avgWeightGradCurvePtr[index] + invWeightMemSize * weightGradCurve;
            avgWeightGradCurveVarPtr[index] = (1.0f - invWeightMemSize) * avgWeightGradCurveVarPtr[index] + invWeightMemSize * weightGradCurve * weightGradCurve;

            index += thisLayerSize; // Increment index for next curvature
        }

        float biasGrad = biasGradPtr[j];
        float biasGradCurve = biasGradCurvePtr[j];

        // Update moving averages for biases
        avgBiasGradPtr[j] = (1.0f - invBiasMemSize) * avgBiasGradPtr[j] + invBiasMemSize * biasGrad;
        avgBiasGradVarPtr[j] = (1.0f - invBiasMemSize) * avgBiasGradVarPtr[j] + invBiasMemSize * biasGrad * biasGrad;
        avgBiasGradCurvePtr[j] = (1.0f - invBiasMemSize) * avgBiasGradCurvePtr[j] + invBiasMemSize * biasGradCurve;
        avgBiasGradCurveVarPtr[j] = (1.0f - invBiasMemSize) * avgBiasGradCurveVarPtr[j] + invBiasMemSize * biasGradCurve * biasGradCurve;
    }
}