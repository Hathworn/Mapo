#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LengthFromElements(float* element1, float* element2, float* output, int count)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread ID directly

    // Ensure threadId is within bounds before performing calculation
    if (threadId < count)
    {
        float e1 = element1[threadId];
        float e2 = element2[threadId];
        output[threadId] = sqrtf(e1 * e1 + e2 * e2);  // Optimize access by reducing multiple indexing
    }
}