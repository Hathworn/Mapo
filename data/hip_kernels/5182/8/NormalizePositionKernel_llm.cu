#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NormalizePositionKernel(float *input, float *normalized, float xMax, float yMax)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Simplified threadId calculation

    if (threadId == 0) // Only first thread performs the normalization
    {
        normalized[0] = input[0] / xMax;
        normalized[1] = input[1] / yMax;
    }
}