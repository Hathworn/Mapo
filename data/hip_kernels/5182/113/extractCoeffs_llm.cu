#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void extractCoeffs(const float *matrices, float *coeffArray, int savedCoeffs, int dimsize) {
    int threadGlobalID = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
                       + threadIdx.z * blockDim.y * blockDim.x 
                       + threadIdx.y * blockDim.x 
                       + threadIdx.x;

    int offsetMatrix = threadGlobalID * dimsize * dimsize;
    int offsetCoeff = threadGlobalID * savedCoeffs;
    int coeffsLeft = savedCoeffs;
    int x, y, y_n = 0, x_n = 1;
    int numberinrow, tmp;

    coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)] = matrices[offsetMatrix];
    coeffsLeft -= 1;

    while (coeffsLeft > 0) {
        int x = x_n, y = y_n;
        
        // Calculate number in row
        numberinrow = (x_n < dimsize - 1) ? x_n + 1 : x_n - (y_n - 1);

        // Use conditional statement to handle even and odd rows
        while (numberinrow > 0 && coeffsLeft > 0) {
            coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)] = matrices[offsetMatrix + x + y * dimsize];
            numberinrow--;
            coeffsLeft--;

            // Efficiently swap x and y for zigzag order
            tmp = x; x = y; y = tmp;

            if (numberinrow % 2 == 0) {
                x--;
                y++;
            }
        }

        // Handle last element in odd row if needed
        if (coeffsLeft > 0 && (x_n >= dimsize - 1 || numberinrow == 0)) {
            coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)] = matrices[offsetMatrix + x + y * dimsize];
            coeffsLeft--;
        }

        // Move to next diagonal
        if (x_n == dimsize - 1) {
            y_n++;
        } else {
            x_n++;
        }
    }
}