#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_getCol_FloatPointer_naive(const float * A, int Acount, int Acols, const float * colId, int empty_par1, int empty_par2, float * out0, int out0count, int out0cols)
{
    // Improved calculation of unique thread id
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread id is within valid range
    if (id < Acount / Acols)
    {
        // Load colId value in shared memory to reduce global memory access
        __shared__ int columnIdx;
        if (threadIdx.x == 0) {
            columnIdx = (int)colId[0];
        }
        __syncthreads();

        // Use shared column index
        out0[id] = A[id*Acols + columnIdx];
    }
}