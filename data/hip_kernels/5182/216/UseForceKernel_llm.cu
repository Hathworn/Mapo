#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UseForceKernel(float *force, float forceFactor, float *pointsCoordinates, int maxCells)
{
    // Calculate a linear thread index across the grid
    int threadId = blockIdx.x * blockDim.x + blockIdx.y * gridDim.x * blockDim.x + threadIdx.x;

    // Ensure threadId does not exceed bounds
    if(threadId < maxCells * 3)
    {
        // Update point coordinates using computed force
        pointsCoordinates[threadId] += forceFactor * force[threadId];
    }
}
```
