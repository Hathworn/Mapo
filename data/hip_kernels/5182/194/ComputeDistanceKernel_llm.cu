#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeDistanceKernel(int inputSize, float *distance, float *dimensionWeight, int maxCells, float *difference)
{
    // Calculate threadId using combined y-dimension of blocks
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    // Check if threadId is within bounds
    if (threadId < maxCells)
    {
        float sum = 0.0f;
        // Cache difference location in shared memory for faster access
        extern __shared__ float sharedDiff[];
        for (int i = threadIdx.x; i < inputSize; i += blockDim.x)
        {
            sharedDiff[i] = difference[threadId * inputSize + i];
        }
        __syncthreads();

        // Compute weighted sum of squared values
        for (int i = 0; i < inputSize; i++)
        {
            float value = sharedDiff[i];
            sum += dimensionWeight[i] * value * value;
        }
        // Update distance array
        distance[threadId] = sqrtf(sum);
    }
}