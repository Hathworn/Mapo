#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PolynomialFunctionKernel(float a3, float a2, float a1, float a0, float* input, float* output, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplify global ID calculation (2D to 1D)

    if (id < size)
    {
        float x = input[id];
        output[id] = ((a3 * x + a2) * x + a1) * x + a0; // Optimize polynomial calculation using Horner's method
    }
}