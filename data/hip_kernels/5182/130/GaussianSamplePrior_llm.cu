#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GaussianSamplePrior(float* input, int inputCount, float* mins, float* maxes, float* randomUniform)
{
    // Calculate unique thread index in grid
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x; 

    if (i < inputCount)
    {
        float diff = maxes[i] - mins[i];
        input[i] = randomUniform[i] * diff + mins[i];
    }
}