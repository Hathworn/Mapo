#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void UpdateCC_XY( float *CCXY, int id_CC, float *XY_tofill, int dim_XY ) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure id is within bounds before processing
    if (id < dim_XY) {
        CCXY[id_CC * dim_XY + id] = XY_tofill[id];
    }
}