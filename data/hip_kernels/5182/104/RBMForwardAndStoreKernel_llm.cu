#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float activationProbability(float x, float sigma)
{
    return 1.0 / (1.0 + expf(-sigma * x));
}

__global__ void RBMForwardAndStoreKernel(float *inputPtr, float *outputPtr, float *weightPtr, float *biasPtr, 
                                         float *storedOutputPtr, float sigma, int prevLayerSize, int thisLayerSize, 
                                         bool useDropout, float *dropoutMask)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index

    if (j < thisLayerSize)
    {
        if (useDropout && !dropoutMask[j]) // Check dropout condition
        {
            outputPtr[j] = 0;
            storedOutputPtr[j] = 0;
        }
        else
        {
            float sum = 0.0f;
            for (int i = 0, index = j; i < prevLayerSize; i++, index += thisLayerSize) // Optimize loop with index increment
            {
                sum += weightPtr[index] * inputPtr[i]; // Accumulate weighted input
            }
            sum += biasPtr[j]; // Add bias
            float result = activationProbability(sum, sigma); // Apply activation function
            outputPtr[j] = result; // Set output value
            storedOutputPtr[j] = result; // Store output value
        }
    }
}