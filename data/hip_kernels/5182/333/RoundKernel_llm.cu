#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RoundKernel(float* input, float* output, int size)
{
    // Calculate unique thread index within the grid
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Use shared memory to reduce global memory accesses
    extern __shared__ float shared_input[];
    
    if (id < size)
    {
        // Load data into shared memory
        shared_input[threadIdx.x] = input[id];
        __syncthreads();
        
        // Perform computation using shared memory
        output[id] = round(shared_input[threadIdx.x]);
    }
}