#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void createCosineMatrix(float* matrix, int xsize) {
    // Calculate global thread ID.
    int threadGlobalID = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
                       + threadIdx.z * blockDim.y * blockDim.x
                       + threadIdx.y * blockDim.x + threadIdx.x;

    // Precompute constants to avoid redundant calculations.
    float invSqrtXsize = 1.0f / sqrtf((float)xsize);
    float sqrt2OverXsize = sqrtf(2.0f / xsize);

    for (int i = 0; i < xsize; i++) {
        if (threadGlobalID == 0) {
            matrix[threadGlobalID + i * xsize] = invSqrtXsize;
        } else {
            // Use precomputed constants.
            float angle = (PI * (2 * i + 1) * threadGlobalID) / (2 * xsize);
            matrix[threadGlobalID + i * xsize] = sqrt2OverXsize * cosf(angle);
        }
    }
}