#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CopyConnectionsCoordinatesKernel(int *connectionMatrix, float *pointsCoordinates, float *vertexData, int *connectionCount, int maxCells)
{
    // Calculate unique thread ID more efficiently
    int threadId = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x; 

    if (threadId < maxCells * maxCells)
    {
        if (connectionMatrix[threadId] == 1)
        {
            int from = threadId / maxCells;
            int to = threadId % maxCells;

            if (to > from)
            {
                // Use atomic operations to ensure correct counter increment
                int connIdx = atomicAdd(&connectionCount[0], 1);

                // Use loop to reduce redundancy in code
                for (int i = 0; i < 3; ++i) 
                {
                    vertexData[connIdx * 6 + i] = pointsCoordinates[from * 3 + i];
                    vertexData[connIdx * 6 + 3 + i] = pointsCoordinates[to * 3 + i];
                }
            }
        }
    }
}