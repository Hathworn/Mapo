#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeCubes2Kernel(float *pointsCoordinates, float *vertexData, int quadOffset, float cubeSide, float *cubeOperation, float *cubeTexCoordinates, int *activityFlag, float textureWidth, int maxCells)
{
    int threadId = blockDim.x * blockIdx.y * gridDim.x // rows preceding current row in grid
                 + blockDim.x * blockIdx.x             // blocks preceding current block
                 + threadIdx.x;

    if (threadId < maxCells * 6)
    {
        int cellId = threadId / 6;
        float fCellId = static_cast<float>(cellId);
        int sideId = threadId % 6;

        float x = pointsCoordinates[cellId * 3];
        float y = pointsCoordinates[cellId * 3 + 1];
        float z = pointsCoordinates[cellId * 3 + 2];

        // Use ternary operation directly without multiplication to save computation
        float halfSide = activityFlag[cellId] ? 0.50f * cubeSide : 0.0f;

        int textureOffset = quadOffset + maxCells * 4 * 6 * 3;
        float textureAbsLength = static_cast<float>(maxCells) * textureWidth;

        #pragma unroll  // Use loop unrolling for performance optimization
        for (int i = 0; i < 4; ++i)
        {
            int vertexIndex = quadOffset + cellId * 72 + 12 * sideId + 3 * i;
            vertexData[vertexIndex]     = x + operationMaskConstant[12 * sideId + 3 * i] * halfSide;
            vertexData[vertexIndex + 1] = y + operationMaskConstant[12 * sideId + 3 * i + 1] * halfSide;
            vertexData[vertexIndex + 2] = z + operationMaskConstant[12 * sideId + 3 * i + 2] * halfSide;

            int texIndex = textureOffset + cellId * 48 + 8 * sideId + 2 * i;
            vertexData[texIndex]     = ((fCellId + cubeTexCoordinatesConstant[sideId * 8 + 2 * i]) * textureWidth) / textureAbsLength;
            vertexData[texIndex + 1] = cubeTexCoordinatesConstant[sideId * 8 + 2 * i + 1];
        }
    }
}