#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GoniometricFunctionKernel(float* input, float* output, const int size, const int type)
{
    // Calculate the unique global thread ID
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    if(id < size)
    {
        float in_val = input[id];  // Load input value once for efficiency

        // Use a lookup table of function pointers for better branching performance
        using func_t = float(*)(float);
        func_t funcs[] = {
            sinf, cosf, tanf, tanhf, sinhf, coshf, asinf, acosf
        };
        
        if (type >= 0 && type <= 7) {
            output[id] = funcs[type](in_val);
        } else if (type == 10) {
            output[id] = atan2f(input[2*id], input[2*id+1]);  // Additional input for atan2f
        }
    }
}