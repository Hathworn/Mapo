#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FinalizeVelocityKernel(float* velocities, float* globalFlow, int inputWidth, int inputHeight)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation
    int gridStride = blockDim.x * gridDim.x; // Grid-stride for processing large arrays
    int size = inputWidth * inputHeight;

    float globalFlowL = sqrtf(globalFlow[0] * globalFlow[0] + globalFlow[1] * globalFlow[1]);

    for (int i = id; i < size; i += gridStride) { // Grid-stride loop for efficient use of resources
        float velocityL = sqrtf(velocities[i] * velocities[i] + velocities[size + i] * velocities[size + i]);

        if (globalFlowL > 0 && velocityL > 0) {
            float dot = (globalFlow[0] * velocities[i] + globalFlow[1] * velocities[size + i]) / (globalFlowL * velocityL);

            if (dot > 0.7) {
                velocities[i] = 0;
                velocities[size + i] = 0;
            }
        }
    }
}