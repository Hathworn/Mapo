#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generateCoefficients(float *chromosomes, const int chromSize, const float* noise, const int population, const int alpha){

int i;

// For up to a 1D grid of 3D blocks...
int threadGlobalID = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
+ threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;

hiprandState st;
hiprand_init((int)noise[threadGlobalID] << threadGlobalID, threadGlobalID * (threadGlobalID == population - 1 ? noise[0] : noise[threadGlobalID]), 0, &st);

if (threadGlobalID > 0){
for (i = 0; i < chromSize; i++){
if (hiprand_uniform(&st) < 0.5){
chromosomes[chromSize*threadGlobalID + i] = hiprand_uniform(&st) *alpha;
}
else{
chromosomes[chromSize*threadGlobalID + i] = -1 * hiprand_uniform(&st) * alpha;
}
}
}
}