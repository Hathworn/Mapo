#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to generate coefficients with optimization for better performance
__global__ void generateCoefficients(float *chromosomes, const int chromSize, const float* noise, const int population, const int alpha){

    // Calculate unique global thread ID
    int threadGlobalID = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize RNG state
    hiprandState st;
    hiprand_init((int)noise[threadGlobalID] << threadGlobalID, threadGlobalID * (threadGlobalID == population - 1 ? noise[0] : noise[threadGlobalID]), 0, &st);

    // Process only valid global thread IDs
    if (threadGlobalID > 0 && threadGlobalID < population * blockDim.x * gridDim.x) {
        // Process chromosome with potential SIMD-friendly loop optimization
        #pragma unroll
        for (int i = 0; i < chromSize; i++){
            float randomValue = hiprand_uniform(&st); // Store RNG result to improve performance
            chromosomes[chromSize * threadGlobalID + i] = (randomValue < 0.5 ? randomValue : -randomValue) * alpha;
        }
    }
}