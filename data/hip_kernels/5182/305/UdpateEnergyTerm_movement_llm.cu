#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UdpateEnergyTerm_movement( float* energy, int energy_dim, int nPatches, float *desc, int desc_dim, int id_desc_move) 
{
    // Optimize block size calculation
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    int idDim = id % energy_dim;
    int idPatch = id / energy_dim;

    // Ensure id is within valid range and access only required memory
    if (id < energy_dim * nPatches && idDim == 1) {
        energy[id] = -desc[idPatch * desc_dim + id_desc_move];
    }
}