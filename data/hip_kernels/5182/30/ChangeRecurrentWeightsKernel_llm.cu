#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ChangeRecurrentWeightsKernel( float *recurrentWeights, float *recurrentWeightDeltas, float *outputWeights, float *outputDeltas, float *recurrentWeightRTRLDerivatives,  float trainingRate, float momentum )
{
    int weightId = blockDim.x * blockIdx.y * gridDim.x + blockDim.x * blockIdx.x + threadIdx.x;

    if (weightId < D_HIDDEN_UNITS * D_HIDDEN_UNITS)
    {
        float gradient = 0;

        // Unrolling the inner loop for better performance
        for (int i = 0; i < D_OUTPUT_UNITS; i++)
        {
            float sum = 0;
            int baseIdx = i * D_HIDDEN_UNITS;
            int rtrlOffset = weightId;

            for (int j = 0; j < D_HIDDEN_UNITS; j++)
            {
                sum += outputWeights[baseIdx + j] * recurrentWeightRTRLDerivatives[j * D_HIDDEN_UNITS * D_HIDDEN_UNITS + rtrlOffset];
            }

            gradient += outputDeltas[i] * sum;
        }

        // Combining operations to reduce memory access
        float weightDelta = trainingRate * gradient + momentum * recurrentWeightDeltas[weightId];
        recurrentWeights[weightId] += weightDelta;
        recurrentWeightDeltas[weightId] = weightDelta;
    }
}