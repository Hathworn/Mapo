#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void detectChanges(float* a, float* b, float* result, int size, float value)
{
// Calculate global thread ID
int threadId = blockIdx.x * blockDim.x + threadIdx.x;

// Ensure thread processes only within bounds
if(threadId < size)
{
    float aValue = a[threadId]; // Load a[threadId] into register
    float bValue = b[threadId]; // Load b[threadId] into register

    // Compare values and store result
    result[threadId] = (aValue > bValue) ? value : (aValue < bValue ? -value : 0.0f);
}
}