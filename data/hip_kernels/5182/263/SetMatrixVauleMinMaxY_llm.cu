#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SetMatrixValueMinMaxY(float* matrix, int cols, int size, int id_min, int id_max, float value)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Get the row index from the thread ID
    int id_row = id / cols;
    
    // Check boundaries and update matrix values
    if (id < size && id_row >= id_min && id_row <= id_max)
    {
        matrix[id] = value;
    }
}