#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AvgCentroidCoordinatesKernel(float *centroidCoordinates, float *pointsWeight, int inputSize, int centroids)
{
    // Calculate global thread index in a simplified way using blockIdx.z dimension
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (threadId < centroids * 2)  // Process only valid threads
    {
        float weight = pointsWeight[threadId / 2];
        // Use ternary for conditional check and assignment
        centroidCoordinates[threadId] = (weight == 0.00f) ? 0.00f : centroidCoordinates[threadId] / weight;
    }
}