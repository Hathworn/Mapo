#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_getRow_FloatId_naive(const float * A, int Acount, int Acols, float * out0, int out0count, int out0cols, const float row_id)
{
    // Calculate global thread ID more efficiently by reducing redundant calculations
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid unnecessary floating-point conversion
    int row_offset = __float2int_rd(row_id) * Acols;

    // Only execute if id is less than Acols
    if (id < Acols)
    {
        out0[id] = A[id + row_offset];
    }
}