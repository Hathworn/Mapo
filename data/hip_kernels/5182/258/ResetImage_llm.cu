#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ResetImage(float* im, int size)
{
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x; // Compute unique thread ID
    if (id < size)
    {
        im[id] = 0.0f; // Reset image value to 0.0f for better precision
    }
}