#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeCubesKernel( float *pointsCoordinates, float *vertexData, int quadOffset, float cubeSide, int *activityFlag, int textureWidth, int maxCells )
{
    int threadId = blockDim.x * blockIdx.y * gridDim.x + blockDim.x * blockIdx.x + threadIdx.x;
    
    if (threadId < maxCells)
    {
        // Precompute common indices and multiplier
        int vertexBaseIdx = quadOffset + threadId * 72;
        int textureBaseIdx = quadOffset + maxCells * 4 * 6 * 3 + threadId * 48;
        float texCoordMultiplier = (float)textureWidth / (float)(maxCells * textureWidth);

        float x = pointsCoordinates[threadId * 3];
        float y = pointsCoordinates[threadId * 3 + 1];
        float z = pointsCoordinates[threadId * 3 + 2];
        float halfSide = (activityFlag[threadId] == 0) ? 0.00f : 0.50f * cubeSide;

        // Helper lambda to set vertex data
        auto setVertexData = [&](int idx, float vx, float vy, float vz) {
            vertexData[vertexBaseIdx + idx] = vx;
            vertexData[vertexBaseIdx + idx + 1] = vy;
            vertexData[vertexBaseIdx + idx + 2] = vz;
        };

        // Helper lambda to set texture data
        auto setTextureData = [&](int idx, float tx, float ty) {
            vertexData[textureBaseIdx + idx] = tx;
            vertexData[textureBaseIdx + idx + 1] = ty;
        };

        // BOTTOM SIDE
        setVertexData(0, x - halfSide, y - halfSide, z + halfSide);
        setVertexData(3, x - halfSide, y - halfSide, z - halfSide);
        setVertexData(6, x + halfSide, y - halfSide, z - halfSide);
        setVertexData(9, x + halfSide, y - halfSide, z + halfSide);

        setTextureData(0, texCoordMultiplier * threadId, 0.00f);
        setTextureData(2, texCoordMultiplier * threadId, 1.00f);
        setTextureData(4, texCoordMultiplier * (threadId + 1), 1.00f);
        setTextureData(6, texCoordMultiplier * (threadId + 1), 0.00f);

        // FRONT SIDE
        setVertexData(12, x - halfSide, y + halfSide, z + halfSide);
        setVertexData(15, x - halfSide, y - halfSide, z + halfSide);
        setVertexData(18, x + halfSide, y - halfSide, z + halfSide);
        setVertexData(21, x + halfSide, y + halfSide, z + halfSide);

        setTextureData(8, texCoordMultiplier * threadId, 0.00f);
        setTextureData(10, texCoordMultiplier * threadId, 1.00f);
        setTextureData(12, texCoordMultiplier * (threadId + 1), 1.00f);
        setTextureData(14, texCoordMultiplier * (threadId + 1), 0.00f);

        // LEFT SIDE
        setVertexData(24, x - halfSide, y + halfSide, z - halfSide);
        setVertexData(27, x - halfSide, y - halfSide, z - halfSide);
        setVertexData(30, x - halfSide, y - halfSide, z + halfSide);
        setVertexData(33, x - halfSide, y + halfSide, z + halfSide);

        setTextureData(16, texCoordMultiplier * threadId, 0.00f);
        setTextureData(18, texCoordMultiplier * threadId, 1.00f);
        setTextureData(20, texCoordMultiplier * (threadId + 1), 1.00f);
        setTextureData(22, texCoordMultiplier * (threadId + 1), 0.00f);

        // BACK SIDE
        setVertexData(36, x - halfSide, y + halfSide, z - halfSide);
        setVertexData(39, x - halfSide, y - halfSide, z - halfSide);
        setVertexData(42, x + halfSide, y - halfSide, z - halfSide);
        setVertexData(45, x + halfSide, y + halfSide, z - halfSide);

        setTextureData(24, texCoordMultiplier * (threadId + 1), 0.00f);
        setTextureData(26, texCoordMultiplier * (threadId + 1), 1.00f);
        setTextureData(28, texCoordMultiplier * threadId, 1.00f);
        setTextureData(30, texCoordMultiplier * threadId, 0.00f);

        // RIGHT SIDE
        setVertexData(48, x + halfSide, y + halfSide, z - halfSide);
        setVertexData(51, x + halfSide, y - halfSide, z - halfSide);
        setVertexData(54, x + halfSide, y - halfSide, z + halfSide);
        setVertexData(57, x + halfSide, y + halfSide, z + halfSide);

        setTextureData(32, texCoordMultiplier * (threadId + 1), 0.00f);
        setTextureData(34, texCoordMultiplier * (threadId + 1), 1.00f);
        setTextureData(36, texCoordMultiplier * threadId, 1.00f);
        setTextureData(38, texCoordMultiplier * threadId, 0.00f);

        // UPPER SIDE
        setVertexData(60, x - halfSide, y + halfSide, z + halfSide);
        setVertexData(63, x - halfSide, y + halfSide, z - halfSide);
        setVertexData(66, x + halfSide, y + halfSide, z - halfSide);
        setVertexData(69, x + halfSide, y + halfSide, z + halfSide);

        setTextureData(40, texCoordMultiplier * threadId, 1.00f);
        setTextureData(42, texCoordMultiplier * threadId, 0.00f);
        setTextureData(44, texCoordMultiplier * (threadId + 1), 0.00f);
        setTextureData(46, texCoordMultiplier * (threadId + 1), 1.00f);
    }
}