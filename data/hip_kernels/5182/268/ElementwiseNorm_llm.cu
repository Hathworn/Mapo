#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ElementwiseNorm(float * A, float *B, int size) {
    // Calculate a single unique index for the thread.
    int id = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x;
    
    // Ensure that the index is within bounds and perform the division.
    if (id < size) {
        A[id] /= B[id];
    }
}