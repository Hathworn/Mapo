#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void OutputDeltaKernel(float *outputDeltas, float *target, float *outputActivations, float *outputActivationDerivatives)
{
    // Calculate unitId using a simplified formula for better readability and performance
    int unitId = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x;
    
    // Check if the unitId is valid and compute the output delta if valid
    if (unitId < D_OUTPUT_UNITS)
    {
        outputDeltas[unitId] = (target[unitId] - outputActivations[unitId]) * outputActivationDerivatives[unitId];
    }
}