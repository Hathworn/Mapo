#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SetVauleInIdxMinMax(float* vector, int id_min, int id_max, float value)
{
    // Calculate the global thread index more efficiently
    int id = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x;
    
    // Use branchless approach for setting value
    if (id >= id_min && id <= id_max) {
        vector[id] = value;
    }
}