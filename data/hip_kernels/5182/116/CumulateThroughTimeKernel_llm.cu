#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CumulateThroughTimeKernel(float* memoryBlocks, int count, int sequenceLength)
{
    int memoryIdx = blockDim.x * blockIdx.y * gridDim.x
                    + blockDim.x * blockIdx.x
                    + threadIdx.x;

    // Optimize by moving invariant calculation out of the loop
    if (memoryIdx < count)
    {
        float cumulatedValue = memoryBlocks[memoryIdx];
        for (size_t i = 1; i < sequenceLength; i++)
        {
            int memoryBlockOffset = i * count;
            cumulatedValue += memoryBlocks[memoryBlockOffset + memoryIdx];
        }
        memoryBlocks[memoryIdx] = cumulatedValue;
    }
}