#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InvolveVector(float* input, float* output, int inputSize)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Calculate thread id in a 1D grid

    if(threadId < inputSize - 1)
    {
        output[threadId + 1] = input[inputSize - threadId - 1]; // Reverse the input directly to position
    }

    if(threadId == 0)
    {
        output[0] = input[0];  // Handle first element separately when threadId is 0
    }
}