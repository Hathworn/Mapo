#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RemoveNodeByUtilityKernel(int *connectionMatrix, int *connectionAge, int *activityFlag, float *utility, float utilityConstant, float *localError, int *neuronAge, float *winningFraction, int *winningCount, float maxError, int maxCells)
{
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x; // Flattened 1D thread index

    if (threadId < maxCells && activityFlag[threadId] == 1 && utility[threadId] > 0.00f)
    {
        if (maxError / utility[threadId] > utilityConstant)
        {
            activityFlag[threadId] = 0;
            localError[threadId] = 0.00f;
            neuronAge[threadId] = 0;
            winningFraction[threadId] = 0.00f;
            winningCount[threadId] = 0;
            utility[threadId] = 0.00f;

            int baseIndex = threadId * maxCells; // Pre-calculate base index for efficiency
            for (int n = 0; n < maxCells; n++)
            {
                connectionMatrix[baseIndex + n] = 0;
                connectionAge[baseIndex + n] = 0;
                connectionMatrix[n * maxCells + threadId] = 0;
                connectionAge[n * maxCells + threadId] = 0;
            }
        }
    }
}