#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SoftmaxKernel(float *outputPtr, float expSum, int layerSize)
{
    // Calculate the global thread ID using 1D grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Only process within bounds
    if (i < layerSize)
    {
        // Optimize division by using multiplication with reciprocal of expSum
        outputPtr[i] *= __frcp_rn(expSum);
    }
}