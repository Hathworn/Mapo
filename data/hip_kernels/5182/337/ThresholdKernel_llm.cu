#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ThresholdKernel(float min, float max, int mode, float* input, float* output, int size, int count)
{
    int id = blockDim.x * blockIdx.y * gridDim.x
           + blockDim.x * blockIdx.x
           + threadIdx.x;

    // Use shared memory efficiently
    __shared__ float delta;

    if(id < size)
    {
        // Initialize delta once per block
        if (threadIdx.x == 0)
            delta = (max - min) / count;
        __syncthreads();

        // Unroll loop for performance
        #pragma unroll
        for (int i = 0; i < count; i++)
            output[i * size + id] = 0;

        int idx;
        float fidx = ((input[id] - min) / delta);

        // Simplify switch-case logic using ternary operator
        idx = (mode == 0)
            ? (int)floor(fmaxf(0, fminf(fidx, count - 1)))
            : ((fidx < 0.0f || fidx >= count) ? -1 : (int)fidx);

        if (idx != -1) // Ensure index is valid for mode 1
            output[idx * size + id] = 1.0f;
    }
}