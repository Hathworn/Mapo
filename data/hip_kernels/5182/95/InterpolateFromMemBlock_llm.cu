#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InterpolateFromMemBlock(float* input1, float* input2, float* output, float* weightMemBlock, int inputSize)
{
    // Calculate the unique thread index
    int threadId = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Load weightMemBlock value once into a register
    float weight = weightMemBlock[0];
    
    if(threadId < inputSize)
    {
        if (weight <= 0) 
        {
            output[threadId] = input1[threadId];
        }
        else if (weight >= 1) 
        {
            output[threadId] = input2[threadId];
        }
        else 
        {
            output[threadId] = (1 - weight) * input1[threadId] + weight * input2[threadId];
        }
    }
}