#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ExpKernel_optimized(const float * A, int Acount, int Acols, float * out0, int out0count)
{
    // Calculate global thread ID using blockIdx and threadIdx
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure id is within the valid range
    if (id < out0count) 
    {
        // Efficient memory access: use shared memory to cache data for faster access
        __shared__ float shared_A[1024]; // Assuming 1024 threads per block; adjust size if needed
        shared_A[threadIdx.x] = A[id];
        __syncthreads();

        // Perform the exp computation
        out0[id] = exp(shared_A[threadIdx.x]);
    }
}