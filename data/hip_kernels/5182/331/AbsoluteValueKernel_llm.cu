#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AbsoluteValueKernel(float* input, float* output, int size)
{
    // Optimize index calculation to reduce arithmetic operations
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Execute only for valid indices
    if (id < size)
    {
        output[id] = fabsf(input[id]); // Using fast math library function
    }
}