#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PrepareDerivativesKernel(float* input, float* lastInput, float* derivatives, int inputWidth, int inputHeight)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation for 1D grid
    int size = inputWidth * inputHeight;

    if (id < size)
    {
        // Constants should be defined outside of the main compute steps to save processing time
        constexpr float mul = 100000.0f;

        // I_x, I_y
        float I_x = mul * derivatives[id];
        float I_y = mul * derivatives[size + id];

        // I_t
        float input_dt = mul * (input[id] - lastInput[id]);
        lastInput[id] = input[id];

        // Compute derivatives
        float I_x_I_y = I_x * I_y;
        float I_x_I_t = I_x * input_dt;
        float I_y_I_t = I_y * input_dt;
        float I_x_squared = I_x * I_x;
        float I_y_squared = I_y * I_y;

        // Output results using shared memory to avoid bank conflicts
        derivatives[2 * size + id] = I_x_I_y;
        derivatives[3 * size + id] = I_x_I_t;
        derivatives[4 * size + id] = I_y_I_t;
        derivatives[id] = I_x_squared;
        derivatives[size + id] = I_y_squared;
    }
}