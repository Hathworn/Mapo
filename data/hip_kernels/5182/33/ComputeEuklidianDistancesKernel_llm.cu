#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeEuklidianDistancesKernel(float *inputImg, int imgWidth, int imgHeight, float *centroidCoordinates, float *distanceMatrix, int centroids, int inputSize)
{
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    // Ensure threadId is within bounds
    if (threadId < inputSize)
    {
        int pointX = threadId % imgWidth;
        int pointY = threadId / imgWidth;

        float X = static_cast<float>(pointX);
        float Y = static_cast<float>(pointY);

        for (int c = 0; c < centroids; c++)
        {
            // Improved memory access pattern for centroid coordinates
            float centroidX = __ldg(&centroidCoordinates[c * 2]);
            float centroidY = __ldg(&centroidCoordinates[c * 2 + 1]);

            float dist = sqrtf((centroidX - X) * (centroidX - X) + (centroidY - Y) * (centroidY - Y));
            // Coalesced memory write to the distance matrix
            distanceMatrix[c * inputSize + threadId] = dist;
        }
    }
}