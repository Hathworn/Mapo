#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeDistanceKernel(float *symbolVectors, float *inputVector, float *distance, int symbolSize, int symbols) {
    // Calculate a unique index for each thread
    int symbolId = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    if (symbolId < symbols) {
        float sum = 0.0f;

        // Use shared memory to optimize access time
        __shared__ float sharedInputVector[1024]; // Assuming symbolSize isn't larger than 1024
        if (threadIdx.x < symbolSize) {
            sharedInputVector[threadIdx.x] = inputVector[threadIdx.x];
        }
        __syncthreads();

        // Main computation using shared memory
        for (int i = 0; i < symbolSize; i++) {
            sum += symbolVectors[symbolId * symbolSize + i] * sharedInputVector[i];
        }

        // Storing result
        distance[symbolId] = sum;
    }
}