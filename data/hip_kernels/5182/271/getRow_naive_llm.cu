#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getRow_optimized(const float * A, float * row_id, float * out, int Acols) {
    // Calculate flattened global index
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    
    // Optimize memory access by calculating row offset only once
    int rowOffset = (int)(*row_id) * Acols;
    
    // Perform data assignment if within valid range
    if (id < Acols) {
        out[id] = A[id + rowOffset];
    }
}