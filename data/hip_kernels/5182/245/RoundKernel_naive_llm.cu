#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RoundKernel_naive(const float * A, int Acount, int Acols, float * out0, int out0count)
{
    // Calculate the global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Only process elements within bounds
    if (id < out0count) {
        out0[id] = roundf(A[id]);  // Use roundf for better performance with float
    }
}