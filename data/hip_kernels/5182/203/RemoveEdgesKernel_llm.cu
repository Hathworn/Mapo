#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RemoveEdgesKernel(int *connection, int *age, int maxAge, int *activityFlag, float *winningFraction, int *winningCount, float *utility, float *localError, int *neuronAge, int maxCells) 
{
    int threadId = blockDim.x * blockIdx.y * gridDim.x // rows preceding current row in grid
                    + blockDim.x * blockIdx.x          // blocks preceding current block
                    + threadIdx.x;

    if (threadId < maxCells) 
    {
        if (activityFlag[threadId] == 1) 
        {
            neuronAge[threadId] += 1;

            int activeConnections = 0;
            int baseIdx = threadId * maxCells;

            for (int c = 0; c < maxCells; c++) 
            {
                int connId = baseIdx + c;
                int connAge = age[connId];

                bool isActive = connection[connId] == 1;
                bool isOld = connAge > maxAge;

                // Conditional reduction of branching by combining conditions
                if (isActive && isOld) 
                {
                    connection[connId] = 0;
                    age[connId] = 0;
                } 
                else if (isActive) 
                {
                    activeConnections++;
                }
            }

            if (activeConnections == 0) 
            {
                activityFlag[threadId] = 0;
                localError[threadId] = 0.00f;
                neuronAge[threadId] = 0;
                winningFraction[threadId] = 0.00f;
                winningCount[threadId] = 0;
                utility[threadId] = 0.00f;
            }
        }
    }
}