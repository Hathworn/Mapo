#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PIDControllerKernel(float* input, float* goal, float* output, float* previousError, float* integral)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified ID calculation.

    if (id < D_COUNT)
    {
        float error = input[id] - goal[id];
        integral[id] = D_INTEGRAL_DECAY * integral[id] + error;
        float derivative = error - previousError[id];

        previousError[id] = error;

        // Merged calculation for output and clamping in one step
        float out = D_OFFSET + D_PROPORTIONAL_GAIN * error 
                    + D_INTEGRAL_GAIN * integral[id] 
                    + D_DERIVATIVE_GAIN * derivative;
        
        // Clamp the output
        output[id] = max(D_MIN_OUTPUT, min(out, D_MAX_OUTPUT));
    }
}