#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SumBasicSymbolsKernel(float *symbolVectors, int symbolOneId, int symbolTwoId, float *result, int symbolSize)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation

    if (threadId < symbolSize)
    {
        // Vector addition for corresponding elements
        result[threadId] = symbolVectors[symbolOneId * symbolSize + threadId] + symbolVectors[symbolTwoId * symbolSize + threadId];
    }
}