#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Copy_A_to_B(float* A, float* B, int size) {
    int id = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x; // Simplified ID calculation
    if (id < size) {
        B[id] = A[id]; // Direct assignment
    }
}