#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MarkCentroidsKernel(float *centroidCoordinates, float *visField, int imgWidth, int imgHeight, int centroids)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread ID
    if (threadId < centroids) {
        int x = lrintf(centroidCoordinates[threadId * 2]);
        int y = lrintf(centroidCoordinates[threadId * 2 + 1]);

        if (x >= 0 && x < imgWidth && y >= 0 && y < imgHeight) { // Ensure coordinates are within bounds
            visField[y * imgWidth + x] = -1.00f;
        }
    }
}