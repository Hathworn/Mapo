#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_getCol_FloatId_naive(const float *A, int Acount, int Acols, float *out0, int out0count, int out0cols, float col_id) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified global thread ID calculation for 1D grid
    if (id < out0count) {
        out0[id] = A[id * Acols + static_cast<int>(col_id)];  // Use static_cast for type conversion
    }
}