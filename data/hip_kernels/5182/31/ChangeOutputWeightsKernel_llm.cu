#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ChangeOutputWeightsKernel( float *outputWeights, float *outputWeightDeltas, float *outputDeltas, float *hiddenActivations,  float trainingRate, float momentum )
{
    int weightId = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation

    if (weightId < D_OUTPUT_UNITS * D_HIDDEN_UNITS)
    {
        int to = weightId / D_HIDDEN_UNITS;
        int from = weightId % D_HIDDEN_UNITS;

        float gradient = outputDeltas[to] * hiddenActivations[from];
        float weightDelta = trainingRate * gradient + momentum * outputWeightDeltas[weightId];
        outputWeightDeltas[weightId] = weightDelta;
        outputWeights[weightId] += weightDelta;
    }
}