#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MaskByNaN(float* inputImage, float* mask, float* outputImage, int count) {
    // Calculate global thread ID more efficiently
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x; 

    // Check if thread ID is within the bounds
    if (id < count) {
        // Apply masking logic
        outputImage[id] = (mask[id] == 0.0f) ? NAN : inputImage[id];
    }
}