#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LSTMDeltaKernel(float *cellStateErrors, float *outputGateDeltas, float *cellStates, float *outputGateActivations, float *outputGateActivationDerivatives, float *deltas, int cellCount, int cellsPerBlock) {
    int memoryBlockId = blockDim.x * blockIdx.y * gridDim.x
                      + blockDim.x * blockIdx.x
                      + threadIdx.x;

    if (memoryBlockId < cellCount / cellsPerBlock) {
        // Efficient use of registers for temporary variables
        float outputGateDeltaSum = 0.0f;
        float memoryBlockActivation = outputGateActivations[memoryBlockId];
        float memoryBlockDerivative = outputGateActivationDerivatives[memoryBlockId];

        int startIdx = memoryBlockId * cellsPerBlock;
        int endIdx = startIdx + cellsPerBlock;

        // Loop unrolling for optimization
        #pragma unroll
        for (int cellId = startIdx; cellId < endIdx; cellId++) {
            float delta = deltas[cellId];
            cellStateErrors[cellId] = memoryBlockActivation * delta;
            outputGateDeltaSum += cellStates[cellId] * delta;
        }

        outputGateDeltas[memoryBlockId] = memoryBlockDerivative * outputGateDeltaSum;
    }
}