#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CopyRectangleCheckBoundsKernel(float *src, int srcOffset, int srcWidth, int srcHeight, int srcRectX, int srcRectY, int rectWidth, int rectHeight, float *dest, int destOffset, int destWidth, int destRectX, int destRectY, float defaultValue)
{
    // Calculate unique thread ID
    int id = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;

    // Pre-calculate total size
    int size = rectWidth * rectHeight;

    // Only process if id is within bounds
    if (id < size) {
        // Determine local x and y positions within the rectangle
        int localX = id % rectWidth;
        int localY = id / rectWidth;

        // Calculate source and destination pixel positions
        int srcPixelX = srcRectX + localX;
        int srcPixelY = srcRectY + localY;
        int destPixelX = destRectX + localX;
        int destPixelY = destRectY + localY;

        // Check if source pixel is within valid bounds
        if (srcPixelX >= 0 && srcPixelX < srcWidth && srcPixelY >= 0 && srcPixelY < srcHeight) {
            dest[destOffset + destPixelX + destPixelY * destWidth] = src[srcOffset + srcPixelX + srcPixelY * srcWidth];
        } else {
            dest[destOffset + destPixelX + destPixelY * destWidth] = defaultValue;
        }
    }
}