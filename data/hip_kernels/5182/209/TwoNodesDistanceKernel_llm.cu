#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TwoNodesDistanceKernel(float *twoNodesDifference, float *twoNodesDistance, int vectorLength)
{
    // Optimize threadId calculation
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure only the first thread computes the distance
    if (threadId == 0)
    {
        float sum = 0.0f;
        // Unroll loop for better performance
        #pragma unroll
        for (int i = 0; i < vectorLength; i++)
        {
            float value = twoNodesDifference[i];
            sum += value * value;
        }
        twoNodesDistance[0] = sqrtf(sum);
    }
}