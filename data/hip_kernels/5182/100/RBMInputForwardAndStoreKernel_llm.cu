#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RBMInputForwardAndStoreKernel(float *inputPtr, float *outputPtr, float *biasPtr, float *storePtr, bool applyBias, int thisLayerSize)
{
    // Calculate unique global thread index
    int i = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    // Ensure within bounds
    if (i < thisLayerSize)
    {
        float result = inputPtr[i];
        // Add bias if applicable
        if (applyBias)
            result += biasPtr[i];
        // Store result in output and store arrays
        outputPtr[i] = result;
        storePtr[i] = result;
    }
}