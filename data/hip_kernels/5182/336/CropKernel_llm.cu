#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CropKernel(float min, float max, float* input, float* output, int size)
{
    // Calculate global index in a more concise way
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a check to avoid accessing beyond the size
    if (id < size)
    {
        // Directly assign the cropped value to the output
        output[id] = fmaxf(fminf(input[id], max), min);
    }
}