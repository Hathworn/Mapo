#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CeilKernel(float* input, float* output, int size)
{
    // Compute the global thread ID more efficiently
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    if(id < size)
    {
        // Using intrinsic function for ceiling
        output[id] = hipCeilf(input[id]);
    }
}