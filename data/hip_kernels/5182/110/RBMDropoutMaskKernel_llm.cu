#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RBMDropoutMaskKernel(float *maskPtr, float dropout, int thisLayerSize) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation

    if (index < thisLayerSize) {
        maskPtr[index] = dropout < maskPtr[index]; // Perform computation
    }
}