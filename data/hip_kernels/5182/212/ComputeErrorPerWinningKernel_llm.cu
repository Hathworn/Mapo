#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeErrorPerWinningKernel(float *localError, int *winningCount, float *errorPerWinning, int *activityFlag, int maxCells)
{
    int threadId = blockDim.x * blockIdx.y * gridDim.x // rows preceding current row in grid
                 + blockDim.x * blockIdx.x             // blocks preceding current block
                 + threadIdx.x;

    if (threadId < maxCells && activityFlag[threadId] == 1) // Combine conditions to minimize branching
    {
        float count = winningCount[threadId];
        errorPerWinning[threadId] = (count != 0) ? localError[threadId] / count : 0.00f; // Use ternary operator for compactness
    }
}