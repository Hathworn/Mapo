#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SetKernel(float *buffer, int offset, float value, int count)
{
    // Use 1D grid and 1D block; simplify threadId computation for better readability
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadId < count)  // Ensure threadId is within bounds
    {
        buffer[offset + threadId] = value; // Directly index with offset
    }
}