#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CopyPointsCoordinatesKernel(float *pointsCoordinates, int *activityFlag, float xNonValid, float yNonValid, float zNonValid, float *dataVertex, int dataVertexOffset, int maxCells)
{
    // Calculate threadId more concisely
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadId < maxCells)
    {
        // Load points coordinates
        float xToCopy = pointsCoordinates[threadId * 3];
        float yToCopy = pointsCoordinates[threadId * 3 + 1];
        float zToCopy = pointsCoordinates[threadId * 3 + 2];

        // Check activityFlag and update coordinates if needed
        if(activityFlag[threadId] == 0)
        {
            xToCopy = xNonValid;
            yToCopy = yNonValid;
            zToCopy = zNonValid;
        }

        // Store results into dataVertex
        int index = dataVertexOffset + threadId * 3;
        dataVertex[index] = xToCopy;
        dataVertex[index + 1] = yToCopy;
        dataVertex[index + 2] = zToCopy;
    }
}