#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CompareVectorsKernel(float* inputOne, float* inputTwo, float* output)
{
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x; // Optimized index calculation

    if (id < gridDim.x * blockDim.x * gridDim.y) { // Check bounds to prevent out-of-bounds access
        if (inputOne[id] != inputTwo[id]) {
            atomicExch(output, 1); // Use atomic operation for thread safety
        }
    }
}