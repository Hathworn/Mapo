#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_PermuteRows(const float * A, int Acount, int Acols, const float * B, int Bcount, int Bcols, float * out0, int out0count, int out0cols)
{
    // Calculate unique thread index
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    // Proceed if within bounds
    if (id < Acount)
    {
        int id_row = id / Acols;
        int id_col = id % Acols;
        int id_rowNew = __fmul_rd(B[id_row], Acols); // Use fast multiply intrinsic

        // Use shared memory for higher speed access if required
        // __shared__ float sharedA[TILE_DIM][TILE_DIM]; // Example: Define TILE_DIM

        out0[id] = A[id_col + id_rowNew];
    }
}