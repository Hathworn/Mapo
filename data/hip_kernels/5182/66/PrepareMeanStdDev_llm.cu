#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PrepareMeanStdDev(float* input, float* delta, int imageWidth, int imageHeight)
{
    // Calculate the unique thread ID and size of the input
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    int size = imageWidth * imageHeight;

    // Ensure the thread is operating within the valid range
    if (id < size)
    {
        // Calculate pixel (x, y) positions
        int px = id % imageWidth;
        int py = id / imageWidth;

        // Precompute common expressions
        float normalizedX = 2.0f * px / imageWidth - 1;
        float normalizedY = 2.0f * py / imageHeight - 1;
        float inputVal = input[id];

        // Calculate mean sums
        delta[id] = inputVal * normalizedX;
        delta[id + size] = inputVal * normalizedY;

        // Calculate variance sums
        delta[id + 2 * size] = inputVal * normalizedX * normalizedX;
        delta[id + 3 * size] = inputVal * normalizedY * normalizedY;
    }
}