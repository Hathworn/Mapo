#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ChangeInputWeightsKernel(float *inputWeights, float *inputWeightDeltas, float *outputWeights, float *outputDeltas, float *inputWeightRTRLDerivatives, float trainingRate, float momentum)
{
    // Calculate flat weightId directly
    int weightId = blockIdx.x * blockDim.x + threadIdx.x + gridDim.x * blockIdx.y * blockDim.x;

    if (weightId < D_HIDDEN_UNITS * D_INPUT_UNITS)
    {
        float gradient = 0.0f;

        // Combine loop iterations using a single kernel calculation for sum
        for (int i = 0; i < D_OUTPUT_UNITS; i++)
        {
            float sum = 0.0f;
            #pragma unroll  // Suggest unrolling for better performance
            for (int j = 0; j < D_HIDDEN_UNITS; j++)
            {
                sum += outputWeights[i * D_HIDDEN_UNITS + j] * inputWeightRTRLDerivatives[j * D_HIDDEN_UNITS * D_INPUT_UNITS + weightId];
            }

            gradient += outputDeltas[i] * sum;
        }

        // Use float operations consistently for delta and weight calculations
        float weightDelta = trainingRate * gradient + momentum * inputWeightDeltas[weightId];
        inputWeightDeltas[weightId] = weightDelta;
        inputWeights[weightId] += weightDelta;
    }
}