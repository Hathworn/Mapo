#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixCopy_naive(const float *A, int Acount, int Acols, float *out0, int out0count)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified ID calculation
    if (id < out0count)
    {
        out0[id] = A[id];
    }
}