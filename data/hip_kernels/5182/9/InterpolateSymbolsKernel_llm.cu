#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InterpolateSymbolsKernel(float *symbolVectors, int symbolOneId, int symbolTwoId, float weightOne, float weightTwo, float *resultSymbol, int symbolSize) {
    // Calculate the unique global thread ID directly
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure threadId is within bounds
    if (threadId < symbolSize) {
        int symbolOneCellId = symbolOneId * symbolSize + threadId;
        int symbolTwoCellId = symbolTwoId * symbolSize + threadId;

        // Perform interpolation
        resultSymbol[threadId] = weightOne * symbolVectors[symbolOneCellId] + weightTwo * symbolVectors[symbolTwoCellId];
    }
}