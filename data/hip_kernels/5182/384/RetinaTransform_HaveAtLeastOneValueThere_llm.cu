#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void EstimateParForSubsample(float* subImageDefs, bool safeBounds, int inputWidth, int inputHeight, int2 & subImg, int & diameterPix) 
{
    diameterPix = (int)(fminf((float)inputWidth, (float)inputHeight) * subImageDefs[2]); // <0,1>
    subImg.x = (int)((float)inputWidth * (subImageDefs[0] + 1) * 0.5f);
    subImg.y = (int)((float)inputHeight * (subImageDefs[1] + 1) * 0.5f);

    int maxDiameter = min(inputWidth - 1, inputHeight - 1);
    diameterPix = max(1, diameterPix);
    diameterPix = min(maxDiameter, diameterPix);

    if (safeBounds) 
    {
        subImg.x = max(subImg.x, 1);
        subImg.y = max(subImg.y, 1);
        subImg.x = min(subImg.x, inputWidth - diameterPix - 1);
        subImg.y = min(subImg.y, inputHeight - diameterPix - 1);
    }
}

__global__ void RetinaTransform_HaveAtLeastOneValueThere(float * subImageDefs, float* input, int inputWidth, int inputHeight, float* output, int outputDataSize, float* retinaMask, int retinaDataSize, int retinaMaskColHint, float* retinaDataInserted) 
{
    int id_retinaPoint = blockDim.x * blockIdx.y * gridDim.x + blockDim.x * blockIdx.x + threadIdx.x;

    int2 subImg;
    int diameterPix;
    bool safeBounds = 0;

    EstimateParForSubsample(subImageDefs, safeBounds, inputWidth, inputHeight, subImg, diameterPix);

    if (id_retinaPoint < outputDataSize) 
    {
        output[id_retinaPoint] = 0; // default value
        float x_mask = (retinaMask[id_retinaPoint * retinaMaskColHint] * diameterPix);
        float y_mask = (retinaMask[id_retinaPoint * retinaMaskColHint + 1] * diameterPix);

        int x = subImg.x + roundf(x_mask); // Use roundf for better precision
        int y = subImg.y + roundf(y_mask); // Use roundf for better precision
        if (x < inputWidth && y < inputHeight && x >= 0 && y >= 0) 
        {
            float val = input[x + y * inputWidth];
            atomicAdd(output + id_retinaPoint, val); // Directly use atomicAdd
            atomicAdd(retinaDataInserted + id_retinaPoint, 1); // Avoid overwriting with direct assignment
        }
    }
}