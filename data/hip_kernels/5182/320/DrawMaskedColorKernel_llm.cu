#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DrawMaskedColorKernel(float *target, int targetWidth, int targetHeight, int inputX, int inputY, float *textureMask, int textureWidth, int textureHeight, float r, float g, float b) 
{
    int id = blockDim.x * blockIdx.y * gridDim.x
           + blockDim.x * blockIdx.x
           + threadIdx.x;

    int targetPixels = targetWidth * targetHeight;
    int texturePixels = textureWidth * textureHeight;

    int idTextureRgb = id / texturePixels;
    int idTexturePixel = id % texturePixels; // Optimized to use modulus directly
    int idTextureY = idTexturePixel / textureWidth;
    int idTextureX = idTexturePixel % textureWidth; // Optimized to use modulus directly

    // Only process RGB channels
    if (idTextureRgb < 3) 
    {
        int offsetX = idTextureX + inputX;
        int offsetY = idTextureY + inputY;

        // Check if within target bounds
        if (offsetX < targetWidth && offsetX >= 0 && offsetY < targetHeight && offsetY >= 0) 
        {
            int tIndex = targetPixels * idTextureRgb + targetWidth * offsetY + offsetX;
            int aIndex = idTexturePixel + 3 * texturePixels; // The A component of the texture
            float a = textureMask[aIndex];

            // Only process if mask allows
            if (a > 0) 
            {
                if (idTextureRgb == 0) 
                {
                    target[tIndex] = r;
                }
                else if (idTextureRgb == 1) 
                {
                    target[tIndex] = g;
                }
                else // idTextureRgb == 2
                {
                    target[tIndex] = b;
                }
            }
        }
    }
}