#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CutSubImageKernel_SingleParams(float *input, float *output, float subImageX, float subImageY, float subImageDiameter, bool safeBounds, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    int size = outputWidth * outputHeight;

    if (id < size)
    {
        // Cache common calculations
        float subImgCX = subImageX; // <-1, 1>
        float subImgCY = subImageY; // <-1, 1>
        float subImgDiameter = subImageDiameter; // <0, 1>
        int maxDiameter = min(inputWidth - 1, inputHeight - 1);
        int diameterPix = min(max(1, (int)(subImgDiameter * maxDiameter)), maxDiameter);

        int subImgX = (int)(inputWidth * (subImgCX + 1) * 0.5f) - diameterPix / 2;
        int subImgY = (int)(inputHeight * (subImgCY + 1) * 0.5f) - diameterPix / 2;

        // Adjust coordinates if safeBounds is true
        if (safeBounds)
        {
            subImgX = max(1, min(subImgX, inputWidth - diameterPix - 1));
            subImgY = max(1, min(subImgY, inputHeight - diameterPix - 1));
        }

        // Calculate position in subimage
        int px = id % outputWidth;
        int py = id / outputWidth;
        float xRatio = (diameterPix - 1) / (float)(outputWidth - 1);
        float yRatio = (diameterPix - 1) / (float)(outputHeight - 1);
        int x = (int)(xRatio * px);
        int y = (int)(yRatio * py);

        // Bounds check for valid access
        if ((x + subImgX) >= 0 && (y + subImgY) >= 0 && (x + subImgX) < inputWidth && (y + subImgY) < inputHeight)
        {
            output[py * outputWidth + px] = input[(y + subImgY) * inputWidth + (x + subImgX)];
        }
    }
}