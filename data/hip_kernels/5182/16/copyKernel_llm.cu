#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyKernel(float* from, float* to, int size)
{
    // Calculate threadId more efficiently using combined block indexing
    int threadId = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.x * gridDim.x);

    if(threadId < size)
    {
        // Utilize coalesced memory access for better performance
        to[threadId] = from[threadId];
    }
}