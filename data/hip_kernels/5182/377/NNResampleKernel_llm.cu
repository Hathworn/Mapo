#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NNResampleKernel(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
    // Calculate global thread index
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    
    // Precompute constants
    float xRatio = (float)(inputWidth - 1) / outputWidth;
    float yRatio = (float)(inputHeight - 1) / outputHeight;
    
    if (id < outputWidth * outputHeight)
    {
        // Compute 2D pixel location from 1D thread index
        int px = id % outputWidth;
        int py = id / outputWidth;
        
        // Compute nearest neighbor positions
        int x = (int)(xRatio * (px + 0.5f));
        int y = (int)(yRatio * (py + 0.5f));
        
        // Assign input pixel to output
        output[py * outputWidth + px] = input[y * inputWidth + x];
    }
}