#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DrawRgbBackgroundKernel(float *target, int inputWidth, int inputHeight, float r, float g, float b)
{
    int column = threadIdx.x + blockDim.x * blockIdx.z;
    if (column >= inputWidth)
        return;

    int row = blockIdx.x;
    int colorChannel = blockIdx.y;

    // linear index with color channel
    int id = colorChannel * (inputWidth * inputHeight) + row * inputWidth + column;

    if (id < 3 * inputWidth * inputHeight) // Simplified boundary check
    {
        float color;
        // Use conditional operator for compactness
        color = (colorChannel == 0) ? r : (colorChannel == 1) ? g : b;
        target[id] = color;
    }
}