#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ResetLayerKernel(float *layer, float value, int count)
{
    // Calculate unique thread ID
    int threadId = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Only apply reset if threadId is within bounds
    if (threadId < count)
    {
        layer[threadId] = value;
    }
}