#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NegativeCorrelationForwardResetKernel(float* outputPtr, int thisLayerSize) {
    // Calculate global thread index
    int j = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Check bounds and reset value
    if (j < thisLayerSize) {
        outputPtr[j] = 0.0f; // Use 0.0f for clarity and consistency with float type
    }
}