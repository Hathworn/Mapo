#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Sum(float * A, float * B, float * C, int size) {
// Calculate unique thread index using 1D block and grid configuration
int id = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x;
// Perform addition only if within bounds
if (id < size) {
    C[id] = A[id] + B[id];
}
}