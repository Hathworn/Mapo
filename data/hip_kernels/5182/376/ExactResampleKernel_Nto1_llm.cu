#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ExactResampleKernel_Nto1(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x; // Simplified id calculation
    int size = outputWidth * outputHeight;

    if (id < size)
    {
        int px = id % outputWidth;
        int py = id / outputWidth;

        int xRatio = inputWidth / outputWidth;
        int yRatio = inputHeight / outputHeight;

        float sum = 0.0f; // Use 0.0f for float precision

        int startX = px * xRatio;
        int startY = py * yRatio;
        
        for (int sx = 0; sx < xRatio; sx++)
        {
            for (int sy = 0; sy < yRatio; sy++)
            {
                int x = startX + sx; // Pre-calculate start indices
                int y = startY + sy;

                sum += input[y * inputWidth + x];
            }
        }

        output[py * outputWidth + px] = sum / (xRatio * yRatio); // Remove redundant float casting
    }
}