#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BilinearResampleKernel(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
    // Calculate global thread id
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    int size = outputWidth * outputHeight;

    if (id < size)
    {
        // Calculate output pixel coordinates
        int px = id % outputWidth;
        int py = id / outputWidth;

        // Precompute ratios
        float xRatio = (float)(inputWidth - 1) / (outputWidth - 1);
        float yRatio = (float)(inputHeight - 1) / (outputHeight - 1);

        // Calculate float coordinates in input
        float x = xRatio * px;
        float y = yRatio * py;

        // Calculate floor and ceil integer indices directly
        int xL = (int)x;
        int xR = min(xL + 1, inputWidth - 1);
        int yT = (int)y;
        int yB = min(yT + 1, inputHeight - 1);

        // Precompute interpolation weights
        float dL = xR - x;
        float dR = x - xL;
        float dT = yB - y;
        float dB = y - yT;

        // Fetch corner pixel values
        float topLeft = input[yT * inputWidth + xL];
        float topRight = input[yT * inputWidth + xR];
        float bottomLeft = input[yB * inputWidth + xL];
        float bottomRight = input[yB * inputWidth + xR];

        // Compute interpolation in x direction
        float iT = topLeft * dL + topRight * dR;
        float iB = bottomLeft * dL + bottomRight * dR;

        // Interpolate in y direction and store the result
        output[py * outputWidth + px] = iT * dT + iB * dB;
    }
}