#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define IDX (blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x)

__device__ void OFConvertXY2AngleSize(float* of, int id, int imageSize, float& of_size, float& of_angle) {
    float2 OF_value;
    
    OF_value.x = of[id];
    OF_value.y = of[id + imageSize];
    
    // Use hypotf for efficient magnitude calculation
    of_size  = hypotf(OF_value.x, OF_value.y);
    // Efficient trigonometric calculation using intrinsics
    of_angle = atan2f(OF_value.x, OF_value.y);
}

__global__ void OFConvert2AngleSize(float* of, int imageSize) {
    int id = IDX;

    if (id < imageSize) {
        float OF_size;
        float OF_angle;
        
        OFConvertXY2AngleSize(of, id, imageSize, OF_size, OF_angle);
        
        of[id] = OF_angle;             // Store calculated angle
        of[id + imageSize] = OF_size;  // Store calculated size
    }
}