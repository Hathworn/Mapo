#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NewNodeConnectionKernel(int f, int q, int r, int *activityFlag, int *connection, int *age, float *localError, float alfa, int maxCells, float errorFraction)
{
    // Calculate thread ID across grid
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    // Process only the first thread
    if (threadId == 0)
    {
        // Update activity flag for node r
        activityFlag[r] = 1;

        // Reset connections and ages for nodes f and q
        connection[q * maxCells + f] = 0;
        age[q * maxCells + f] = 0;
        connection[f * maxCells + q] = 0;
        age[f * maxCells + q] = 0;

        // Set connections and ages for nodes q and r, f and r
        connection[q * maxCells + r] = 1;
        age[q * maxCells + r] = 0;
        connection[r * maxCells + q] = 1;
        age[r * maxCells + q] = 0;
        connection[f * maxCells + r] = 1;
        age[f * maxCells + r] = 0;
        connection[r * maxCells + f] = 1;
        age[r * maxCells + f] = 0;

        // Update local error values for nodes q, f, and r
        localError[q] -= alfa * localError[q];
        localError[f] -= alfa * localError[f];
        localError[r] = errorFraction * (localError[q] + localError[f]);
    }
}