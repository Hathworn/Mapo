#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AdaptWinningFractionKernel( int s1, float *winningFraction, int *winningCount, float bParam, int maxCells )
{
    // Calculate global thread ID
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure threadId is within valid range
    if(threadId < maxCells)
    {
        // Update winningFraction
        float adjustment = (threadId == s1) ? 1.0f : 0.0f; // Simplify expression
        winningFraction[threadId] += bParam * (adjustment - winningFraction[threadId]);

        // Update winningCount
        winningCount[threadId] += (threadId == s1); // Simplify expression
    }
}