#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SumSymbolsKernel(float *symbolOne, float *symbolTwo, float *result, int symbolSize)
{
    // Calculate global thread index using standard 1D grid-stride loop pattern
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;

    for (int i = threadId; i < symbolSize; i += gridSize) 
    {
        result[i] = symbolOne[i] + symbolTwo[i]; // Perform element-wise addition
    }
}