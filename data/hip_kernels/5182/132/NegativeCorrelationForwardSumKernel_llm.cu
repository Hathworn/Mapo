#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NegativeCorrelationForwardSumKernel( float* inputPtr, float* outputPtr, int thisLayerSize )
{
    // Calculate index j using efficient block indexing technique
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundaries and perform addition
    if (j < thisLayerSize)
    {
        outputPtr[j] += inputPtr[j];
    }
}