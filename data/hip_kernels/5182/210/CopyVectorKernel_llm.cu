#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CopyVectorKernel(float *from, int fromOffset, float *to, int toOffset, int vectorSize)
{
    // Calculate the overall thread index directly in the flattened grid
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < vectorSize)
    {
        // Perform the copy operation
        to[threadId + toOffset] = from[threadId + fromOffset];
    }
}