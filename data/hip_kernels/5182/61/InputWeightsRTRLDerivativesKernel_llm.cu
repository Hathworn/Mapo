#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InputWeightsRTRLDerivativesKernel(float *input, float *hiddenActivationDerivatives, float *recurrentWeights, float *inputWeightRTRLDerivatives, float *previousInputWeightRTRLDerivatives)
{
    // Optimize calculation of global unique thread index
    int partialId = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    if (partialId < D_HIDDEN_UNITS * D_HIDDEN_UNITS * D_INPUT_UNITS)
    {
        // Precompute indices derived from partialId
        int unitId = partialId / (D_HIDDEN_UNITS * D_INPUT_UNITS);
        int weightId = partialId % (D_HIDDEN_UNITS * D_INPUT_UNITS);
        int to = weightId / D_INPUT_UNITS;
        int from = weightId % D_INPUT_UNITS;

        // Use register to store the sum
        float sum = 0;

        // Use constant stride access pattern for memory coalescing
        for (int i = 0; i < D_HIDDEN_UNITS; i++)
        {
            sum += recurrentWeights[unitId * D_HIDDEN_UNITS + i] * previousInputWeightRTRLDerivatives[i * (D_HIDDEN_UNITS * D_INPUT_UNITS) + weightId];
        }

        // Directly compute final result
        inputWeightRTRLDerivatives[partialId] = hiddenActivationDerivatives[unitId] * ((unitId == to ? input[from] : 0) + sum);
    }
}