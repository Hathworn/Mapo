#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FillAdjacencyMatrix(float* adj_mat, float* maskBuffer, int size, int cols, int rows, int Nsegs) {
    // Calculate global index once
    int idx = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    int icol = idx % cols;
    int irow = idx / cols;

    // Ensure idx is within bounds and inside the effective area
    if (idx < size && icol < cols - 2 && icol > 1 && irow < rows - 2 && irow > 1) {
        int seg_id1 = maskBuffer[idx];
        
        // Check adjacency with right neighbor
        if (seg_id1 != maskBuffer[idx + 1]) {
            int seg_id2 = maskBuffer[idx + 1];
            // Set adjacency in both directions
            adj_mat[seg_id2 + seg_id1 * Nsegs] = 1;
            adj_mat[seg_id1 + seg_id2 * Nsegs] = 1;
        }
        // Check adjacency with upper neighbor
        else if (seg_id1 != maskBuffer[idx - cols]) {
            int seg_id2 = maskBuffer[idx - cols];
            // Set adjacency in both directions
            adj_mat[seg_id2 + seg_id1 * Nsegs] = 1;
            adj_mat[seg_id1 + seg_id2 * Nsegs] = 1;
        }
    }
}