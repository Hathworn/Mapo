#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function using 2D grid and reducing multiplication
__global__ void PolynomialFunctionKernel_Double(float a3, float a2, float a1, float a0, double* input, double* output, int size)
{
    int id = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x; // Calculate global thread ID

    if (id < size)
    {
        double x = input[id];
        double x2 = x * x; // Pre-compute x squared
        output[id] = (a3 * x2 + a2) * x2 + a1 * x + a0; // Minimize calculations by rearranging
    }
}