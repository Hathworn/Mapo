#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InvertValuesKernel(float *input, float* outputs, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Process only valid indices
    if (id < size)
    {
        // Compute inverted value
        outputs[id] = 1.00f - input[id];
    }
}