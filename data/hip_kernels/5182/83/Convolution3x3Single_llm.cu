#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float mulWithKernel(int x, int y, int kx, int ky, float* input, int width, int height)
{
    int px = min(max(x, 0), width - 1);
    int py = min(max(y, 0), height - 1);
    return D_KERNEL[3 * (ky + 1) + kx + 1] * input[py * width + px];
}

__global__ void Convolution3x3Single(float* input, float* output, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure the thread is within the output boundaries
    if(x < width && y < height)
    {
        float result = 0;

        // Unroll the loop to minimize loop overhead
        result += mulWithKernel(x - 1, y - 1, -1, -1, input, width, height);
        result += mulWithKernel(x - 1, y    , -1,  0, input, width, height);
        result += mulWithKernel(x - 1, y + 1, -1,  1, input, width, height);

        result += mulWithKernel(x, y - 1, 0, -1, input, width, height);
        result += mulWithKernel(x, y    , 0,  0, input, width, height);
        result += mulWithKernel(x, y + 1, 0,  1, input, width, height);

        result += mulWithKernel(x + 1, y - 1, 1, -1, input, width, height);
        result += mulWithKernel(x + 1, y    , 1,  0, input, width, height);
        result += mulWithKernel(x + 1, y + 1, 1,  1, input, width, height);

        output[y * width + x] = result;
    }
}