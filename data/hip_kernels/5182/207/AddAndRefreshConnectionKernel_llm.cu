#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AddAndRefreshConnectionKernel(int node1, int node2, int *activityFlag, int *connection, int *age, int maxCells)
{
    const int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId == 0) // Use only the first thread to perform the operation
    {
        // Mark nodes as active
        activityFlag[node1] = 1;
        activityFlag[node2] = 1;

        // Establish and refresh connection, set age to 0
        const int idx1 = node1 * maxCells + node2;
        const int idx2 = node2 * maxCells + node1;
        connection[idx1] = 1;
        age[idx1] = 0;
        connection[idx2] = 1;
        age[idx2] = 0;
    }
}