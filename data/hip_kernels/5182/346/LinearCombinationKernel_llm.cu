#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LinearCombinationKernel(float *input1, float input1_coeff, int input1_start_index, float *input2, float input2_coeff, int input2_start_index, float *output, int output_start_index, int size)
{
    // Improved method to calculate global thread ID for better readability
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure valid indexing of the output array
    if(id < size)
    {
        output[output_start_index + id] = input1_coeff * input1[input1_start_index + id] + input2_coeff * input2[input2_start_index + id];
    }
}