#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UniformNormalDistribution(float *from, float *to, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified id calculation

    if (id < size)
    {
        // Combined calculations into fewer operations
        float tmp = normcdf(from[id] * rsqrtf((float)size)); 

        to[id] = (tmp - 0.5f) * 2.0f;
    }
}