#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SumNewCentroidCoordinatesKernel(float *input, int imgWidth, int imgHeight, float *centroidCoordinates, int *nearestCentroid, float *pointsWeight, int inputSize)
{
    int threadId = blockDim.x * blockIdx.x + threadIdx.x; // Calculate thread ID within the grid

    if(threadId < inputSize)
    {
        int pointX = threadId % imgWidth;
        int pointY = threadId / imgWidth;

        float X = static_cast<float>(pointX);
        float Y = static_cast<float>(pointY);

        int centroidId = nearestCentroid[threadId];

        float weight = input[threadId];

        // Use shared memory to reduce atomic operations
        __shared__ float sharedCentroidCoords[2];
        __shared__ float sharedPointsWeight;

        if (threadIdx.x == 0) {
            sharedCentroidCoords[0] = 0.0f;
            sharedCentroidCoords[1] = 0.0f;
            sharedPointsWeight = 0.0f;
        }
        __syncthreads();

        // Accumulate the values in shared memory
        atomicAdd(&sharedCentroidCoords[0], weight * X);
        atomicAdd(&sharedCentroidCoords[1], weight * Y);
        atomicAdd(&sharedPointsWeight, weight);
        __syncthreads();

        // Only one thread writes back the accumulated results to global memory
        if (threadIdx.x == 0) {
            atomicAdd(&centroidCoordinates[centroidId * 2], sharedCentroidCoords[0]);
            atomicAdd(&centroidCoordinates[centroidId * 2 + 1], sharedCentroidCoords[1]);
            atomicAdd(&pointsWeight[centroidId], sharedPointsWeight);
        }
    }
}