#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getRow_IntId_naive(const float * A, int row_id, float * out, int Acols) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified and corrected index calculation
    if (id < Acols) {
        out[id] = A[id + row_id * Acols];
    }
}