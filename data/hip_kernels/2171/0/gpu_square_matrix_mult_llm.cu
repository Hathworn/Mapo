#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_square_matrix_mult(float *d_a, float *d_b, float *d_result, int n)
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;

    // Loop over submatrices of d_a and d_b
    for (int sub = 0; sub < gridDim.x; ++sub)
    {
        // Load tile_a from d_a
        int a_idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        tile_a[threadIdx.y][threadIdx.x] = (a_idx < n * n) ? d_a[a_idx] : 0;

        // Load tile_b from d_b
        int b_idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        tile_b[threadIdx.y][threadIdx.x] = (b_idx < n * n) ? d_b[b_idx] : 0;

        __syncthreads(); // Wait for all threads to complete loading

        // Compute partial product for tile
        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x]; // Matrix multiplication
        }
        __syncthreads(); // Ensure all threads complete computation
    }

    // Write final result to global memory
    if (row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}