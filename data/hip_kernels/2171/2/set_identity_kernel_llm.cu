#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_identity_kernel(float *a, int m, int n) {
    int idx = blockIdx.y * blockDim.y + threadIdx.y;
    int idy = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by ensuring computation only for relevant thread indices
    if (idx < m && idy < n) {
        int index = idx * n + idy; // Compute element index once
        a[index] = (idx == idy) ? 1.0f : 0.0f; // Simplify conditional logic
    }
}