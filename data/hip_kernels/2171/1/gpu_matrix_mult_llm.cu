#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_matrix_mult(float *a, float *b, float *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;  // Use float for sum

    if (col < k && row < m)
    {
        // Unroll loop for better performance
        for (int i = 0; i < n; i += 4)
        {
            sum += a[row * n + i] * b[i * k + col];
            if (i + 1 < n) sum += a[row * n + i + 1] * b[(i + 1) * k + col];
            if (i + 2 < n) sum += a[row * n + i + 2] * b[(i + 2) * k + col];
            if (i + 3 < n) sum += a[row * n + i + 3] * b[(i + 3) * k + col];
        }
        c[row * k + col] = sum;
    }
}