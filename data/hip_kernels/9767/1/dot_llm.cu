#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(float *a, float *b, float *c, int threadperblock, int max) {
    __shared__ float cache[ThreadPerBlock];

    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int cacheindex = threadIdx.x;
    float temp = 0;

    // Loop unrolling for better performance 
    for (int i = tid; i < max; i += gridDim.x * blockDim.x) {
        temp += a[i] * b[i];  // Accumulating directly inside the loop
    }

    cache[cacheindex] = temp;
    __syncthreads();

    // Use a for-loop with bitwise operation for reduction
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cacheindex < i) {
            cache[cacheindex] += cache[cacheindex + i];
        }
        __syncthreads();
    }

    // Only one thread writes the result back to global memory
    if (cacheindex == 0) {
        c[blockIdx.x] = cache[0];
    }
}