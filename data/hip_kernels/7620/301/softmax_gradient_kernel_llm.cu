#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel( const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // Partial sum for the inner products
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Single thread reduction to compute the sum across threads
    if (idx == 0) {
        tmp = 0.0f;
        for (int i = 0; i < blockDim.x; ++i) {
            tmp += reduction_buffer[i];
        }
        reduction_buffer[0] = tmp;
    }
    __syncthreads();

    // Compute gradient using the calculated sum
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}