#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int CHUNKS = 64;
const int GENERATIONS = 10;

const int CHECK_VALUES_EVERY = 50000;
const int SHOW_ALL_VALUES = 0;
const int SKIP_CUDA_DEVICE = false;

const int ISLANDS_PER_ROW = 4;
const int GENOME_LENGTH = 4;
const int BLOCKS_PER_ROW = 4;
const int ISLAND_POPULATION = 100;
const int SELECTION_COUNT = 80;
const float MUTATION_CHANCE = 0.8;
const int ITEMS_MAX_WEIGHT = 5;
const int ITEMS_MAX_VALUE = 20;
const int ITEMS_MAX = 20;

__global__ void init(unsigned int seed, hiprandState_t* states) {
  // Simplify state initialization using linear indexing
  int id = blockDim.x * blockIdx.x + threadIdx.x; // Compute unique index for each thread
  hiprand_init(seed, id, 0, &states[id]); // Initialize with unique sequence number
}