#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 8 // size of the matrices to be multiplied
#define TILE_WIDTH 4 // size of the tiles

__global__ void MatrixMulKernel(float* d_M, float* d_N, float* d_P, int Width) {
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;  
    int by = blockIdx.y;
    int tx = threadIdx.x; 
    int ty = threadIdx.y;

    // Identify the row and column of the Pd element to work on
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float Pvalue = 0.0f; // Use float for consistency and efficiency
    // Loop over the Md and Nd tiles required to compute the Pd element
    for (int m = 0; m < Width / TILE_WIDTH; ++m) {
        // Collaborative loading of Md and Nd tiles into shared memory
        if (Row < Width && (m*TILE_WIDTH+tx) < Width)
            ds_M[ty][tx] = d_M[Row * Width + m * TILE_WIDTH + tx];
        else
            ds_M[ty][tx] = 0.0f;

        if ((m*TILE_WIDTH+ty) < Width && Col < Width)
            ds_N[ty][tx] = d_N[(m * TILE_WIDTH + ty) * Width + Col];
        else
            ds_N[ty][tx] = 0.0f;
        
        __syncthreads();
        
        // Compute tiles
        for (int k = 0; k < TILE_WIDTH; ++k)
            Pvalue += ds_M[ty][k] * ds_N[k][tx];
        
        __syncthreads();
    }
    
    // Write the computed value to the matrix P
    if (Row < Width && Col < Width)
        d_P[Row * Width + Col] = Pvalue;
}