#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void iterate(float* originalMatrixD, float* solutionD, int originalMatrixWidth, int startingIndex) {
    // Use linear shared memory array for simplicity
    __shared__ float originalMatrixDS[TILE_WIDTH * TILE_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int currentMatrixIndex = blockId * (blockDim.x * blockDim.y) + (ty * blockDim.x) + tx + startingIndex;

    // Load data into shared memory
    originalMatrixDS[ty * TILE_WIDTH + tx] = originalMatrixD[currentMatrixIndex];

    // Ensure all threads are synchronized after loading shared memory
    __syncthreads();

    // Determine if on edge
    bool onEdge = (currentMatrixIndex % originalMatrixWidth == 0) || 
                  (currentMatrixIndex % originalMatrixWidth == originalMatrixWidth - 1) ||
                  (currentMatrixIndex < originalMatrixWidth) || 
                  (currentMatrixIndex >= (originalMatrixWidth * (originalMatrixWidth - 1)));

    float replaceAmount;
    if (onEdge) {
        // Keep original value on edges
        replaceAmount = originalMatrixDS[ty * TILE_WIDTH + tx];
    } else {
        // Efficiently calculate neighbors' indices
        int idxLeft = (tx > 0) ? tx - 1 : -1;
        int idxRight = (tx < TILE_WIDTH - 1) ? tx + 1 : -1;
        float top = originalMatrixD[currentMatrixIndex - originalMatrixWidth];
        float bottom = originalMatrixD[currentMatrixIndex + originalMatrixWidth];
        
        float left = (idxLeft != -1) ? originalMatrixDS[ty * TILE_WIDTH + idxLeft] : originalMatrixD[currentMatrixIndex - 1];
        float right = (idxRight != -1) ? originalMatrixDS[ty * TILE_WIDTH + idxRight] : originalMatrixD[currentMatrixIndex + 1];

        // Calculate new value based on neighbors
        replaceAmount = (left + right + top + bottom) / 4;
    }

    // Store calculated value in solution matrix
    solutionD[currentMatrixIndex] = replaceAmount;
}