#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADSPBLK 1024
#define THREADSPSM 2048
#define TILE_WIDTH 32
#define TOTAL_ITERATIONS 50

int main_n;

__global__ void iterate(float* originalMatrixD, float* solutionD, int originalMatrixWidth, int startingIndex) {
    // Use linear indexing for shared memory to improve memory coalescing
    __shared__ float originalMatrixDS[TILE_WIDTH * TILE_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int blockId = blockIdx.x + blockIdx.y * gridDim.x;

    int currentMatrixIndex = blockId * (blockDim.x * blockDim.y) +
                            (threadIdx.y * blockDim.x) + threadIdx.x;
    currentMatrixIndex += startingIndex;

    originalMatrixDS[ty * TILE_WIDTH + tx] = originalMatrixD[currentMatrixIndex];

    // Sync up w/ shared data set up
    __syncthreads();

    float replaceAmount;
    bool onEdge = false;
    int XEdgeCheckMod = currentMatrixIndex % originalMatrixWidth;

    // Check edges using condensed logic for better readability
    onEdge = (XEdgeCheckMod == 0 || XEdgeCheckMod == originalMatrixWidth - 1 || 
              currentMatrixIndex < originalMatrixWidth || 
              currentMatrixIndex >= (originalMatrixWidth * originalMatrixWidth - originalMatrixWidth));

    if (onEdge) {
        replaceAmount = originalMatrixDS[ty * TILE_WIDTH + tx];
    } else {
        // Use registers to store data read from global memory
        float top = originalMatrixD[currentMatrixIndex - originalMatrixWidth];
        float bottom = originalMatrixD[currentMatrixIndex + originalMatrixWidth];
        float left;
        float right;

        // Read boundaries conditionally to minimize branch divergence
        left = (tx == 0) ? originalMatrixD[currentMatrixIndex - 1] : originalMatrixDS[ty * TILE_WIDTH + tx - 1];
        right = (tx == TILE_WIDTH - 1) ? originalMatrixD[currentMatrixIndex + 1] : originalMatrixDS[ty * TILE_WIDTH + tx + 1];

        replaceAmount = (left + right + top + bottom) * 0.25f; // Use multiplication instead of division for efficiency
    }

    solutionD[currentMatrixIndex] = replaceAmount;
}