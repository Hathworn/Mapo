#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generateGaussian_kernel(float* og, float delta, int radius)
{
    int x = threadIdx.x - radius;
    float delta_squared = 2 * delta * delta; // Precompute delta squared for reuse
    og[threadIdx.x] = __expf(-(x * x) / delta_squared); // Use the precomputed value
}