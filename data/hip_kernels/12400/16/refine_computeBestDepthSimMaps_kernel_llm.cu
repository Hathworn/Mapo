#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float refineDepthSubPixel(const float3& depths, const float3& sims)
{
    float outDepth = -1.0f;

    // Normalize similarity measures
    float simM1 = (sims.x + 1.0f) * 0.5f;
    float simP1 = (sims.z + 1.0f) * 0.5f;
    float sim1 = (sims.y + 1.0f) * 0.5f;

    // Check if refinement is needed
    if((simM1 > sim1) && (simP1 > sim1))
    {
        float dispStep = -(simP1 - simM1) / (2.0f * (simP1 + simM1 - 2.0f * sim1));
        float floatDepthM1 = depths.x;
        float floatDepthP1 = depths.z;

        // Calculate refined depth
        float b = (floatDepthP1 + floatDepthM1) * 0.5f;
        float a = b - floatDepthM1;
        outDepth = a * dispStep + b;
    }

    return outDepth;
}

__global__ void refine_computeBestDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float3* isims, int isims_p, float3* idpts, int idpts_p, int width, int height, float simThr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < width) && (y < height))
    {
        float3 depths = idpts[y * idpts_p + x];
        float3 sims = isims[y * isims_p + x];

        // Calculate the best depth and similarity
        float outDepth = (sims.x < sims.y) ? depths.x : depths.y;
        float outSim = (sims.x < sims.y) ? sims.x : sims.y;
        if (sims.z < outSim)
        {
            outDepth = depths.z;
            outSim = sims.z;
        }

        // Attempt a sub-pixel refinement of the depth
        float refinedDepth = refineDepthSubPixel(depths, sims);
        if(refinedDepth > 0.0f)
        {
            outDepth = refinedDepth;
        }

        // Store results with thresholding
        osim[y * osim_p + x] = (outSim < simThr ? outSim : 1.0f);
        odpt[y * odpt_p + x] = (outSim < simThr ? outDepth : -1.0f);
    }
}