#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void refine_fuseThreeDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float* isimLst, int isimLst_p, float* idptLst, int idptLst_p, float* isimAct, int isimAct_p, float* idptAct, int idptAct_p, int width, int height, float simThr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if((x < width) && (y < height))
    {
        float dpts[3];
        float sims[3];

        // Load points and similarity values only once
        dpts[0] = odpt[y * odpt_p + x];
        dpts[1] = idptLst[y * idptLst_p + x];
        dpts[2] = idptAct[y * idptAct_p + x];

        sims[0] = osim[y * odpt_p + x];
        sims[1] = isimLst[y * isimLst_p + x];
        sims[2] = isimAct[y * isimAct_p + x];

        // Pre-compute initial values
        float outDpt = dpts[0];
        float outSim = simThr;

        // Optimize by reducing redundant use of ternary operators
        #pragma unroll
        for(int i = 1; i < 3; ++i) {
            if(sims[i] < outSim) {
                outDpt = dpts[i];
                outSim = sims[i];
            }
        }

        // Assign the computed results to output
        osim[y * osim_p + x] = outSim;
        odpt[y * odpt_p + x] = outDpt;
    };
}