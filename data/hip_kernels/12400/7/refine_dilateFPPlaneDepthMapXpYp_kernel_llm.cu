#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refine_dilateFPPlaneDepthMapXpYp_kernel(float* __restrict__ fpPlaneDepthMap, int fpPlaneDepthMap_p, const float* __restrict__ maskMap, int maskMap_p, int width, int height, int xp, int yp, float fpPlaneDepth)
{
    // Calculate global thread coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds and perform operation only for valid coordinates
    if((x + xp >= 0) && (y + yp >= 0) && (x + xp < width) && (y + yp < height) && (x < width) && (y < height))
    {
        // Read once from global memory
        float depth = maskMap[y * maskMap_p + x];
        if(depth > 0.0f)
        {
            // Write the computed depth value to global memory
            fpPlaneDepthMap[(y + yp) * fpPlaneDepthMap_p + (x + xp)] = fpPlaneDepth;
        }
    }
}