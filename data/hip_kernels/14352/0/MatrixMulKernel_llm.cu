#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulKernel(int * _matrixA, int * _matrixB, int * _result, int _width)
{
    // Cache elements in shared memory for reuse
    __shared__ int sharedA[32][32];
    __shared__ int sharedB[32][32];

    // 2D thread ID
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x * 32;
    int by = blockIdx.y * 32;

    int value = 0;

    // Loop over tiles
    for (int m = 0; m < _width; m += 32) {
        // Load elements into shared memory
        sharedA[ty][tx] = _matrixA[(by + ty) * _width + (m + tx)];
        sharedB[ty][tx] = _matrixB[(m + ty) * _width + (bx + tx)];

        __syncthreads(); // Synchronize to make sure the data is available

        // Perform partial matrix multiplication
        for (int k = 0; k < 32; k++) {
            value += sharedA[ty][k] * sharedB[k][tx];
        }

        __syncthreads(); // Synchronize before loading the next tile
    }

    // Write result to global memory
    _result[(by + ty) * _width + (bx + tx)] = value;
}