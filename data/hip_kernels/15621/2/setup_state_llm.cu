#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Launch thread initializes its own random seed
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}