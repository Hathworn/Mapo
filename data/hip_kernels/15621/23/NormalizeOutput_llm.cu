#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Calculate global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize by using warp-level operations
    int warpSize = warpSize; // Typically 32
    idx *= 3;
    for (int i = idx; i < num_elements * 3; i += blockDim.x * gridDim.x * 3) {
        // Use temporary variables and warp operations to reduce redundant calculations
        int64_t temp_batch = batch_index;
        int64_t temp_class = class_index;
        int64_t temp_orig = static_cast<int64_t>(original[i / 3]);
        
        to_normalize[i] = temp_batch;
        to_normalize[i + 1] = temp_class;
        to_normalize[i + 2] = temp_orig;
    }
}