#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "normal_eqs_disparity_multicam_GPU.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
float *d_CD = NULL;
hipMalloc(&d_CD, XSIZE*YSIZE);
float *d_disparity_compact = NULL;
hipMalloc(&d_disparity_compact, XSIZE*YSIZE);
float4 *d_Zbuffer_normals_compact = NULL;
hipMalloc(&d_Zbuffer_normals_compact, XSIZE*YSIZE);
int *d_ind_disparity_Zbuffer = NULL;
hipMalloc(&d_ind_disparity_Zbuffer, XSIZE*YSIZE);
const float *d_focal_length = NULL;
hipMalloc(&d_focal_length, XSIZE*YSIZE);
const float *d_nodal_point_x = NULL;
hipMalloc(&d_nodal_point_x, XSIZE*YSIZE);
const float *d_nodal_point_y = NULL;
hipMalloc(&d_nodal_point_y, XSIZE*YSIZE);
const float *d_baseline = NULL;
hipMalloc(&d_baseline, XSIZE*YSIZE);
const int *d_n_cols = NULL;
hipMalloc(&d_n_cols, XSIZE*YSIZE);
const int *d_n_values_disparity = NULL;
hipMalloc(&d_n_values_disparity, XSIZE*YSIZE);
const int *d_start_ind_disparity = NULL;
hipMalloc(&d_start_ind_disparity, XSIZE*YSIZE);
const int *d_pixel_ind_offset = NULL;
hipMalloc(&d_pixel_ind_offset, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
normal_eqs_disparity_multicam_GPU<<<gridBlock,threadBlock>>>(d_CD,d_disparity_compact,d_Zbuffer_normals_compact,d_ind_disparity_Zbuffer,d_focal_length,d_nodal_point_x,d_nodal_point_y,d_baseline,d_n_cols,d_n_values_disparity,d_start_ind_disparity,d_pixel_ind_offset);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
normal_eqs_disparity_multicam_GPU<<<gridBlock,threadBlock>>>(d_CD,d_disparity_compact,d_Zbuffer_normals_compact,d_ind_disparity_Zbuffer,d_focal_length,d_nodal_point_x,d_nodal_point_y,d_baseline,d_n_cols,d_n_values_disparity,d_start_ind_disparity,d_pixel_ind_offset);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
normal_eqs_disparity_multicam_GPU<<<gridBlock,threadBlock>>>(d_CD,d_disparity_compact,d_Zbuffer_normals_compact,d_ind_disparity_Zbuffer,d_focal_length,d_nodal_point_x,d_nodal_point_y,d_baseline,d_n_cols,d_n_values_disparity,d_start_ind_disparity,d_pixel_ind_offset);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}