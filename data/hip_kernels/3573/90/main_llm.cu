#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "normal_eqs_disparity_multicam_GPU_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *d_CD = NULL;
hipMalloc(&d_CD, XSIZE*YSIZE);
float *d_disparity_compact = NULL;
hipMalloc(&d_disparity_compact, XSIZE*YSIZE);
float4 *d_Zbuffer_normals_compact = NULL;
hipMalloc(&d_Zbuffer_normals_compact, XSIZE*YSIZE);
int *d_ind_disparity_Zbuffer = NULL;
hipMalloc(&d_ind_disparity_Zbuffer, XSIZE*YSIZE);
const float *d_focal_length = NULL;
hipMalloc(&d_focal_length, XSIZE*YSIZE);
const float *d_nodal_point_x = NULL;
hipMalloc(&d_nodal_point_x, XSIZE*YSIZE);
const float *d_nodal_point_y = NULL;
hipMalloc(&d_nodal_point_y, XSIZE*YSIZE);
const float *d_baseline = NULL;
hipMalloc(&d_baseline, XSIZE*YSIZE);
const int *d_n_cols = NULL;
hipMalloc(&d_n_cols, XSIZE*YSIZE);
const int *d_n_values_disparity = NULL;
hipMalloc(&d_n_values_disparity, XSIZE*YSIZE);
const int *d_start_ind_disparity = NULL;
hipMalloc(&d_start_ind_disparity, XSIZE*YSIZE);
const int *d_pixel_ind_offset = NULL;
hipMalloc(&d_pixel_ind_offset, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
normal_eqs_disparity_multicam_GPU<<<gridBlock, threadBlock>>>(d_CD,d_disparity_compact,d_Zbuffer_normals_compact,d_ind_disparity_Zbuffer,d_focal_length,d_nodal_point_x,d_nodal_point_y,d_baseline,d_n_cols,d_n_values_disparity,d_start_ind_disparity,d_pixel_ind_offset);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
normal_eqs_disparity_multicam_GPU<<<gridBlock, threadBlock>>>(d_CD,d_disparity_compact,d_Zbuffer_normals_compact,d_ind_disparity_Zbuffer,d_focal_length,d_nodal_point_x,d_nodal_point_y,d_baseline,d_n_cols,d_n_values_disparity,d_start_ind_disparity,d_pixel_ind_offset);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
normal_eqs_disparity_multicam_GPU<<<gridBlock, threadBlock>>>(d_CD,d_disparity_compact,d_Zbuffer_normals_compact,d_ind_disparity_Zbuffer,d_focal_length,d_nodal_point_x,d_nodal_point_y,d_baseline,d_n_cols,d_n_values_disparity,d_start_ind_disparity,d_pixel_ind_offset);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}