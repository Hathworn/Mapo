#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normal_eqs_disparity_multicam_GPU( float *d_CD, float *d_disparity_compact, float4 *d_Zbuffer_normals_compact, int *d_ind_disparity_Zbuffer, const float *d_focal_length, const float *d_nodal_point_x, const float *d_nodal_point_y, const float *d_baseline, const int *d_n_cols, const int *d_n_values_disparity, const int *d_start_ind_disparity, const int *d_pixel_ind_offset) {
    int n_val_accum = gridDim.x * blockDim.x;

    int n_disparity = d_n_values_disparity[blockIdx.y];
    int n_accum = (int)ceilf((float)n_disparity / (float)n_val_accum);
    int start_ind = d_start_ind_disparity[blockIdx.y];

    float f = d_focal_length[blockIdx.y];
    float ox = d_nodal_point_x[blockIdx.y];
    float oy = d_nodal_point_y[blockIdx.y];
    float b = d_baseline[blockIdx.y];
    int n_cols = d_n_cols[blockIdx.y];
    int pixel_ind_offset = d_pixel_ind_offset[blockIdx.y];

    __shared__ float smem_CD[27];
    if (threadIdx.x < 27) smem_CD[threadIdx.x] = 0.0f;
    __syncthreads();

    for (int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x;
         in_ind < blockDim.x * (blockIdx.x + 1) * n_accum; in_ind += blockDim.x) {
        if (in_ind < n_disparity) {
            float disp = d_disparity_compact[in_ind + start_ind];
            float4 tmp = d_Zbuffer_normals_compact[in_ind + start_ind];
            float Zbuffer = tmp.x;
            float nx = tmp.y;
            float ny = tmp.z;
            float nz = tmp.w;

            int pixel_ind = d_ind_disparity_Zbuffer[in_ind + start_ind] - pixel_ind_offset;

            float y = floorf(__fdividef((float)pixel_ind, n_cols));
            float x = (float)pixel_ind - y * n_cols;

            x = __fdividef((x - ox), f);
            y = -__fdividef((y - oy), f);

            float Zd = -(f * b) / disp;
            float Xd = x * Zd;
            float Yd = y * Zd;

            float Zm = Zbuffer;
            float Xm = x * Zm;
            float Ym = y * Zm;

            atomicAdd(&smem_CD[0], nx * nx);
            atomicAdd(&smem_CD[1], nx * ny);
            atomicAdd(&smem_CD[2], nx * nz);
            atomicAdd(&smem_CD[3], Ym * nx * nz - Zm * nx * ny);
            atomicAdd(&smem_CD[4], Zm * (nx * nx) - Xm * nx * nz);
            atomicAdd(&smem_CD[5], -Ym * (nx * nx) + Xm * nx * ny);
            atomicAdd(&smem_CD[6], ny * ny);
            atomicAdd(&smem_CD[7], ny * nz);
            atomicAdd(&smem_CD[8], -Zm * (ny * ny) + Ym * ny * nz);
            atomicAdd(&smem_CD[9], -Xm * ny * nz + Zm * nx * ny);
            atomicAdd(&smem_CD[10], Xm * (ny * ny) - Ym * nx * ny);
            atomicAdd(&smem_CD[11], nz * nz);
            atomicAdd(&smem_CD[12], Ym * (nz * nz) - Zm * ny * nz);
            atomicAdd(&smem_CD[13], -Xm * (nz * nz) + Zm * nx * nz);
            atomicAdd(&smem_CD[14], Xm * ny * nz - Ym * nx * nz);
            atomicAdd(&smem_CD[15], (Ym * Ym) * (nz * nz) + (Zm * Zm) * (ny * ny) -
                     Ym * Zm * ny * nz * 2.0f);
            atomicAdd(&smem_CD[16], -Xm * Ym * (nz * nz) - (Zm * Zm) * nx * ny + Xm * Zm * ny * nz +
                     Ym * Zm * nx * nz);
            atomicAdd(&smem_CD[17], -Xm * Zm * (ny * ny) - (Ym * Ym) * nx * nz + Xm * Ym * ny * nz +
                     Ym * Zm * nx * ny);
            atomicAdd(&smem_CD[18], (Xm * Xm) * (nz * nz) + (Zm * Zm) * (nx * nx) -
                     Xm * Zm * nx * nz * 2.0f);
            atomicAdd(&smem_CD[19], -Ym * Zm * (nx * nx) - (Xm * Xm) * ny * nz + Xm * Ym * nx * nz +
                     Xm * Zm * nx * ny);
            atomicAdd(&smem_CD[20], (Xm * Xm) * (ny * ny) + (Ym * Ym) * (nx * nx) -
                     Xm * Ym * nx * ny * 2.0f);
            atomicAdd(&smem_CD[21], Xd * (nx * nx) - Xm * (nx * nx) + Yd * nx * ny - Ym * nx * ny +
                     Zd * nx * nz - Zm * nx * nz);
            atomicAdd(&smem_CD[22], Yd * (ny * ny) - Ym * (ny * ny) + Xd * nx * ny - Xm * nx * ny +
                     Zd * ny * nz - Zm * ny * nz);
            atomicAdd(&smem_CD[23], Zd * (nz * nz) - Zm * (nz * nz) + Xd * nx * nz - Xm * nx * nz +
                     Yd * ny * nz - Ym * ny * nz);
            atomicAdd(&smem_CD[24], -Yd * Zm * (ny * ny) + Ym * Zd * (nz * nz) + Ym * Zm * (ny * ny) -
                     Ym * Zm * (nz * nz) - (Ym * Ym) * ny * nz + (Zm * Zm) * ny * nz +
                     Xd * Ym * nx * nz - Xm * Ym * nx * nz - Xd * Zm * nx * ny +
                     Yd * Ym * ny * nz + Xm * Zm * nx * ny - Zd * Zm * ny * nz);
            atomicAdd(&smem_CD[25], Xd * Zm * (nx * nx) - Xm * Zd * (nz * nz) - Xm * Zm * (nx * nx) +
                     Xm * Zm * (nz * nz) + (Xm * Xm) * nx * nz - (Zm * Zm) * nx * nz -
                     Xd * Xm * nx * nz - Xm * Yd * ny * nz + Xm * Ym * ny * nz +
                     Yd * Zm * nx * ny - Ym * Zm * nx * ny + Zd * Zm * nx * nz);
            atomicAdd(&smem_CD[26], -Xd * Ym * (nx * nx) + Xm * Yd * (ny * ny) + Xm * Ym * (nx * nx) -
                     Xm * Ym * (ny * ny) - (Xm * Xm) * nx * ny + (Ym * Ym) * nx * ny +
                     Xd * Xm * nx * ny - Yd * Ym * nx * ny + Xm * Zd * ny * nz -
                     Xm * Zm * ny * nz - Ym * Zd * nx * nz + Ym * Zm * nx * nz);
        }
    }
    __syncthreads();

    int out_ind = 27 * n_val_accum * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;

    if (threadIdx.x < 27) d_CD[out_ind] = smem_CD[threadIdx.x];
}