#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertFloatToRGBA_kernel(uchar4 *out_image, const float *in_image, int width, int height, float lowerLim, float upperLim) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify index calculation
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    uchar4 temp;
    if (x < width && y < height) {
        float val = in_image[y * width + x];

        if (!isfinite(val)) {
            temp = make_uchar4(255, 255, 255, 255);  // Use make_uchar4 for initialization
        } else {
            val = (val - lowerLim) / (upperLim - lowerLim);  // Combine operations
            float r, g, b;
            if (val < 0.25f) {
                r = 0;
                g = 4.0f * val;
                b = 1.0f;
            } else if (val < 0.5f) {
                r = 0;
                g = 1.0f;
                b = 1.0f + 4.0f * (0.25f - val);
            } else if (val < 0.75f) {
                r = 4.0f * (val - 0.5f);
                g = 1.0f;
                b = 0;
            } else {
                r = 1.0f;
                g = 1.0f + 4.0f * (0.75f - val);
                b = 0;
            }
            temp = make_uchar4(255.0f * r, 255.0f * g, 255.0f * b, 255);  // Simplify assignment
        }
        out_image[y * width + x] = temp;
    }
}