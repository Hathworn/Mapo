#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample_ind_and_labels_GPU(int *d_ind_sub, const int *d_ind, unsigned int *d_label_sub, const unsigned int *d_label, int n_out, float inv_sub_factor) {

    // Cached value of global index for better readability and efficiency
    unsigned int ind_out = blockIdx.x * blockDim.x + threadIdx.x; 

    if (ind_out < n_out) {
        // Calculate the input index using multiplication directly 
        // and cast it as int. Avoid float arithmetic when possible.
        int ind_in = static_cast<int>(ind_out * inv_sub_factor);
        
        // Directly access and assign the subsampled indexes and labels
        d_ind_sub[ind_out] = d_ind[ind_in]; 
        d_label_sub[ind_out] = d_label[ind_in];
    }
}