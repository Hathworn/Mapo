#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void createAnaglyph_kernel(uchar4 *out_image, const float *left_image, const float *right_image, int width, int height, int pre_shift) {
    // Calculate x and y using more efficient grid calculations
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x_right = x - pre_shift;
    uchar4 temp;

    if (x < width && y < height) {
        int idx = y * width + x; // Calculate index once
        temp.x = left_image[idx]; // Load left image pixel

        // Check bounds of shifted x_right
        if (x_right > 0 && x_right < width) {
            temp.y = right_image[y * width + x_right]; // Load right image pixel
            temp.z = temp.y; // Copy to temp.z
        } else {
            temp.y = 0;
            temp.z = 0;
        }

        temp.w = 255; // Set alpha channel
        out_image[idx] = temp; // Store result
    }
}