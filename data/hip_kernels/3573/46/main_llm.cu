#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "convert2DVectorToAngleMagnitude_kernel_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    uchar4 *d_angle_image = NULL;
hipMalloc(&d_angle_image, XSIZE*YSIZE);
uchar4 *d_magnitude_image = NULL;
hipMalloc(&d_magnitude_image, XSIZE*YSIZE);
float *d_vector_X = NULL;
hipMalloc(&d_vector_X, XSIZE*YSIZE);
float *d_vector_Y = NULL;
hipMalloc(&d_vector_Y, XSIZE*YSIZE);
int width = XSIZE;
int height = YSIZE;
float lower_ang = 1;
float upper_ang = 1;
float lower_mag = 1;
float upper_mag = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
convert2DVectorToAngleMagnitude_kernel<<<gridBlock, threadBlock>>>(d_angle_image,d_magnitude_image,d_vector_X,d_vector_Y,width,height,lower_ang,upper_ang,lower_mag,upper_mag);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
convert2DVectorToAngleMagnitude_kernel<<<gridBlock, threadBlock>>>(d_angle_image,d_magnitude_image,d_vector_X,d_vector_Y,width,height,lower_ang,upper_ang,lower_mag,upper_mag);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
convert2DVectorToAngleMagnitude_kernel<<<gridBlock, threadBlock>>>(d_angle_image,d_magnitude_image,d_vector_X,d_vector_Y,width,height,lower_ang,upper_ang,lower_mag,upper_mag);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}