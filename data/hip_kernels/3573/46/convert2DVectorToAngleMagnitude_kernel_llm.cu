#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to convert 2D vector to angle and magnitude
__global__ void convert2DVectorToAngleMagnitude_kernel(uchar4 *d_angle_image, uchar4 *d_magnitude_image, float *d_vector_X, float *d_vector_Y, int width, int height, float lower_ang, float upper_ang, float lower_mag, float upper_mag) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Use standard index calculation
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    uchar4 temp_angle, temp_magnitude;

    if (x < width && y < height) {
        float vector_X = d_vector_X[y * width + x]; // Simplify array access
        float vector_Y = d_vector_Y[y * width + x];

        // Compute angle and magnitude
        float angle = atan2f(vector_Y, vector_X);
        float magnitude = hypotf(vector_X, vector_Y); // Use hypotf for magnitude

        // First draw unmatched pixels in white
        if (!isfinite(magnitude)) {
            temp_angle = make_uchar4(255, 255, 255, 255); // Use make_uchar4 for initialization
            temp_magnitude = make_uchar4(255, 255, 255, 255);
        } else {
            // Rescale angle and magnitude from [lower,upper] to [0,1] and convert to RGBA jet colorspace
            angle = (angle - lower_ang) / (upper_ang - lower_ang);

            float r = 1.0f, g = 1.0f, b = 1.0f;

            // Optimize color calculation using fewer branches
            if (angle < 0.25f) {
                r = 0;
                g = angle * 4.0f;
            } else if (angle < 0.5f) {
                r = 0;
                b = 1.0f + 4.0f * (0.25f - angle);
            } else if (angle < 0.75f) {
                r = 4.0f * (angle - 0.5f);
                b = 0;
            } else {
                g = 1.0f + 4.0f * (0.75f - angle);
                b = 0;
            }

            temp_angle = make_uchar4(static_cast<unsigned char>(255.0f * r), 
                                     static_cast<unsigned char>(255.0f * g), 
                                     static_cast<unsigned char>(255.0f * b), 
                                     255);

            magnitude = (magnitude - lower_mag) / (upper_mag - lower_mag);

            r = 1.0f, g = 1.0f, b = 1.0f;

            if (magnitude < 0.25f) {
                r = 0;
                g = magnitude * 4.0f;
            } else if (magnitude < 0.5f) {
                r = 0;
                b = 1.0f + 4.0f * (0.25f - magnitude);
            } else if (magnitude < 0.75f) {
                r = 4.0f * (magnitude - 0.5f);
                b = 0;
            } else {
                g = 1.0f + 4.0f * (0.75f - magnitude);
                b = 0;
            }

            temp_magnitude = make_uchar4(static_cast<unsigned char>(255.0f * r), 
                                      static_cast<unsigned char>(255.0f * g), 
                                      static_cast<unsigned char>(255.0f * b), 
                                      255);
        }

        d_angle_image[y * width + x] = temp_angle; // Simplified index calculations
        d_magnitude_image[y * width + x] = temp_magnitude;
    }
}