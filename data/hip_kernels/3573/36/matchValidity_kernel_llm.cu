#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matchValidity_kernel(float *d_flow, float *d_disparity, int n_cols, int n_rows) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure threads are within bounds using guard clauses
    if (x >= n_cols || y >= n_rows) return; 

    unsigned int ind = x + y * n_cols;
    // Use a single line to reduce code branching
    if (!(isfinite(d_flow[ind]) && isfinite(d_disparity[ind]))) {
        float nan_value = nanf("");
        d_flow[ind] = nan_value;
        d_flow[ind + n_cols * n_rows] = nan_value;
        d_disparity[ind] = nan_value;
    }
}