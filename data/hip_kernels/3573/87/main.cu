#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "normal_eqs_flow_GPU.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *d_CO = NULL;
hipMalloc(&d_CO, XSIZE*YSIZE);
const float2 *d_flow_compact = NULL;
hipMalloc(&d_flow_compact, XSIZE*YSIZE);
const float *d_Zbuffer_flow_compact = NULL;
hipMalloc(&d_Zbuffer_flow_compact, XSIZE*YSIZE);
const int *d_ind_flow_Zbuffer = NULL;
hipMalloc(&d_ind_flow_Zbuffer, XSIZE*YSIZE);
float fx = 1;
float fy = 1;
float ox = 1;
float oy = 1;
int n_rows = 1;
int n_cols = 1;
const int *d_n_values_flow = NULL;
hipMalloc(&d_n_values_flow, XSIZE*YSIZE);
const int *d_start_ind_flow = NULL;
hipMalloc(&d_start_ind_flow, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
normal_eqs_flow_GPU<<<gridBlock, threadBlock>>>(d_CO,d_flow_compact,d_Zbuffer_flow_compact,d_ind_flow_Zbuffer,fx,fy,ox,oy,n_rows,n_cols,d_n_values_flow,d_start_ind_flow);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
normal_eqs_flow_GPU<<<gridBlock, threadBlock>>>(d_CO,d_flow_compact,d_Zbuffer_flow_compact,d_ind_flow_Zbuffer,fx,fy,ox,oy,n_rows,n_cols,d_n_values_flow,d_start_ind_flow);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
normal_eqs_flow_GPU<<<gridBlock, threadBlock>>>(d_CO,d_flow_compact,d_Zbuffer_flow_compact,d_ind_flow_Zbuffer,fx,fy,ox,oy,n_rows,n_cols,d_n_values_flow,d_start_ind_flow);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}