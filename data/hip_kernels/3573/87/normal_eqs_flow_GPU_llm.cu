#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normal_eqs_flow_GPU(float *d_CO, const float2 *d_flow_compact, const float *d_Zbuffer_flow_compact, const int *d_ind_flow_Zbuffer, float fx, float fy, float ox, float oy, int n_rows, int n_cols, const int *d_n_values_flow, const int *d_start_ind_flow) {

    int n_val_accum = gridDim.x * blockDim.x; 
    int n_flow = d_n_values_flow[blockIdx.y];
    int n_accum = (int)ceilf((float)n_flow / (float)n_val_accum);
    int start_ind = d_start_ind_flow[blockIdx.y];

    // initialize accumulators
    float A[23] = {0.0f};

    for (int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x;
         in_ind < blockDim.x * (blockIdx.x + 1) * n_accum; in_ind += blockDim.x) {

        if (in_ind < n_flow) {
            // fetch flow and Zbuffer from global memory
            float2 u = d_flow_compact[in_ind + start_ind];
            float disp = __fdividef(1.0f, d_Zbuffer_flow_compact[in_ind + start_ind]);

            // compute coordinates
            int pixel_ind = d_ind_flow_Zbuffer[in_ind + start_ind];
            bool is_ar_flow = (pixel_ind >= (n_rows * n_cols));
            pixel_ind -= (int)is_ar_flow * n_rows * n_cols;

            float y = floorf(__fdividef((float)pixel_ind, n_cols));
            float x = (float)pixel_ind - y * n_cols;

            x = x - ox;
            y = y - oy;

            /************************/
            /* evaluate constraints */
            /************************/

            // unique values A-matrix
            A[0] += (disp * disp * fx * fx);
            A[1] += (-disp * disp * x * fx);
            A[2] += (-disp * x * y);
            A[3] += (disp * fx * fx + disp * x * x);
            A[4] += (-disp * y * fx);
            A[5] += (-disp * disp * y * fy);
            A[6] += (-disp * fy * fy - disp * y * y);
            A[7] += (disp * x * fy);
            A[8] += (disp * disp * x * x + disp * disp * y * y);
            A[9] += (disp * x * x * y / fx + disp * y * fy + disp * y * y * y / fy);
            A[10] += (-disp * x * fx - disp * x * x * x / fx - disp * x * y * y / fy);
            A[11] += (x * x * y * y / (fx * fx) + fy * fy + 2.0f * y * y +
                      y * y * y * y / (fy * fy));
            A[12] += (-2.0f * x * y - x * x * x * y / (fx * fx) -
                      x * y * y * y / (fy * fy));
            A[13] += (x * y * y / fx - x * fy - x * y * y / fy);
            A[14] += (fx * fx + 2.0f * x * x + x * x * x * x / (fx * fx) +
                      x * x * y * y / (fy * fy));
            A[15] += (-y * fx - x * x * y / fx + x * x * y / fy);
            A[16] += (x * x + y * y);

            // B-vector
            A[17] += (disp * u.x * fx);
            A[18] += (disp * u.y * fy);
            A[19] += (-disp * x * u.x - disp * y * u.y);
            A[20] += (-x * y * u.x / fx - u.y * fy - u.y * y * y / fy);
            A[21] += (u.x * fx + x * x * u.x / fx + x * y * u.y / fy);
            A[22] += (-y * u.x + x * u.y);
        }
    }

    /**************************/
    /* write out accumulators */
    /**************************/

    int out_ind = 23 * n_val_accum * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;

    #pragma unroll
    for (int i = 0; i < 23; i++) {
        d_CO[out_ind + i * n_val_accum] = A[i];
    }
}