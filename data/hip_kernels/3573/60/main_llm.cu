#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "colorDistDiff_kernel_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    uchar4 *out_image = NULL;
hipMalloc(&out_image, XSIZE*YSIZE);
const float *disparity = NULL;
hipMalloc(&disparity, XSIZE*YSIZE);
int disparity_pitch = 2;
const float *disparity_prior = NULL;
hipMalloc(&disparity_prior, XSIZE*YSIZE);
int width = XSIZE;
int height = YSIZE;
float f = 1;
float b = 2;
float ox = 1;
float oy = 1;
float dist_thres = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
colorDistDiff_kernel<<<gridBlock, threadBlock>>>(out_image,disparity,disparity_pitch,disparity_prior,width,height,f,b,ox,oy,dist_thres);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
colorDistDiff_kernel<<<gridBlock, threadBlock>>>(out_image,disparity,disparity_pitch,disparity_prior,width,height,f,b,ox,oy,dist_thres);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
colorDistDiff_kernel<<<gridBlock, threadBlock>>>(out_image,disparity,disparity_pitch,disparity_prior,width,height,f,b,ox,oy,dist_thres);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}