#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertPitchedFloatToRGBA_kernel(uchar4 *out_image, const float *in_image, int width, int height, int pitch, float lowerLim, float upperLim) {
    // Calculate flattened thread index
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        uchar4 temp;
        float val = *((float *)((char *)in_image + y * pitch) + x);

        // Check if value is finite, otherwise set white
        if (!isfinite(val)) {
            temp = make_uchar4(255, 255, 255, 255);
        } else {
            // Rescale value from [lowerLim, upperLim] to [0,1]
            val = (val - lowerLim) / (upperLim - lowerLim);

            float r = 1.0f, g = 1.0f, b = 1.0f;
            if (val < 0.25f) {
                r = 0;
                g = 4.0f * val;
            } else if (val < 0.5f) {
                r = 0;
                b = 1.0 + 4.0f * (0.25f - val);
            } else if (val < 0.75f) {
                r = 4.0f * (val - 0.5f);
                b = 0;
            } else {
                g = 1.0f + 4.0f * (0.75f - val);
                b = 0;
            }
            // Set RGBA values
            temp = make_uchar4(__float2int_rn(255.0f * r), __float2int_rn(255.0f * g), __float2int_rn(255.0f * b), 255);
        }
        // Assign pixel value in output image
        out_image[y * width + x] = temp;
    }
}