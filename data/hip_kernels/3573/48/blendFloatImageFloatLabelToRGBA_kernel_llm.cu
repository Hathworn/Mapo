#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blendFloatImageFloatLabelToRGBA_kernel(uchar4 *out_image, const float *in_image, const float *label, int width, int height, float lowerLim, float upperLim) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = y * width + x;
        unsigned char img = static_cast<unsigned char>(0.5f * in_image[index] + 128.0f);
        float val = label[index];

        uchar4 temp;
        if (!isfinite(val) || (val < lowerLim) || (val > upperLim)) {
            // Set all out-of-range or invalid values to white
            temp = make_uchar4(img, img, img, 255);
        } else {
            // Blend valid values
            temp = make_uchar4(static_cast<unsigned char>(0.6f * img),
                               static_cast<unsigned char>(0.6f * img),
                               img, 255);
        }

        out_image[index] = temp;
    }
}
```
