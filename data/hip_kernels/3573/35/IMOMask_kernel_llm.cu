#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void IMOMask_kernel(float *d_IMOMask, float *d_IMO, const float *d_disparity, float offset, int n_cols, int n_rows) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < n_cols && y < n_rows) { // use logical AND for better readability
        unsigned int ind = x + y * n_cols;
        // Perform the operation only when needed
        if (d_IMOMask[ind] == 0.0f) {
            d_IMO[ind] = nanf("");
        }
    }
}