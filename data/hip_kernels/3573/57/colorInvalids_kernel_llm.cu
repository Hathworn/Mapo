#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void colorInvalids_kernel(uchar4 *out_image, const float *in_image, int width, int height) {
    // Using blockIdx.x * blockDim.x + threadIdx.x for index calculation (replacing deprecated __mul24)
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int ind = y * width + x; // Compute image index
        uchar4 temp = out_image[ind];
        float value = in_image[ind];

        if (!isfinite(value)) { // color invalid entries
            temp.x *= 0.5f; // Scale red component
            temp.y *= 0.5f; // Scale green component
        }

        out_image[ind] = temp; // Write back to output image
    }
}