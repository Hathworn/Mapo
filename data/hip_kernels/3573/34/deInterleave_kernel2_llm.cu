#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void deInterleave_kernel2(float *d_X_out, float *d_Y_out, char *d_XY_in, int pitch_out, int pitch_in, int width, int height) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < width) && (y < height)) { // Ensure both conditions must be true
        // Calculate source and destination pointers directly with proper types
        float2 *data = (float2 *)(d_XY_in + y * pitch_in) + x;
        float *destX = (float *)((char *)d_X_out + y * pitch_out) + x;
        float *destY = (float *)((char *)d_Y_out + y * pitch_out) + x;
        
        // Fetch and store data
        *destX = data->x;
        *destY = data->y;
    }
}