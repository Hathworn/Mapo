#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertDepthImageToMeter_kernel(float *d_depth_image_meter, const unsigned int *d_depth_image_millimeter, int n_rows, int n_cols) {

    // Calculate index using flattened 2D grid and block indexes
    const int ind = blockIdx.y * blockDim.y * n_cols + blockIdx.x * blockDim.x + threadIdx.y * n_cols + threadIdx.x;

    // Check if index is within image bounds
    if (ind < n_rows * n_cols) {
        unsigned int depth = d_depth_image_millimeter[ind];
        // Perform conversion from millimeters to meters
        d_depth_image_meter[ind] = (depth == 4294967295) ? nanf("") : (float)depth / 1000.0f;
    }
}