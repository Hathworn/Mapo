#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_normal_eqs_64_GPU(float *d_C_reduced, float *d_C, int gridDim_x_normal_equations) {

    int tid = threadIdx.x;
    int bx = blockIdx.x;

    // put data in shared memory
    int ind = blockIdx.y * gridDim.x * gridDim_x_normal_equations * 64 +
              bx * gridDim_x_normal_equations * 64 + tid;

    __shared__ float DATA[64];

    // load and sum the first 20 elements
    float tmp = 0.0f;
    for (int i = 0; i < gridDim_x_normal_equations; i++)
        tmp += d_C[ind + i * 64];
    DATA[tid] = tmp;

    __syncthreads(); // ensure reading stage has finished

    // optimized reduction using warp shuffle instructions
    for (int stride = 32; stride > 0; stride /= 2) {
        tmp += __shfl_down_sync(0xFFFFFFFF, tmp, stride);
    }
    
    // write results
    if (tid == 0) {
        d_C_reduced[blockIdx.y * gridDim.x + bx] = tmp;
    }
}