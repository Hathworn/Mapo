#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce_normal_eqs_64_mult_constr_GPU(float *d_C_reduced, const float *d_C, int gridDim_x_normal_equations, int n_constraints) {
    int constraint_ind = blockIdx.x * 4 + threadIdx.y;

    if (constraint_ind < n_constraints) {
        int tid = 64 * threadIdx.y + threadIdx.x;
        int ind = blockIdx.y * n_constraints * gridDim_x_normal_equations * 64 +
                  constraint_ind * gridDim_x_normal_equations * 64 + threadIdx.x;

        __shared__ float DATA[64 * 4];

        // Load data into shared memory and perform reduction
        float tmp = 0.0f;
        for (int i = 0; i < gridDim_x_normal_equations; i++)
            tmp += d_C[ind + i * 64];
        DATA[tid] = tmp;

        __syncthreads();

        // Use warp shuffle for efficient reduction
        for (int offset = 32; offset > 0; offset /= 2) {
            tmp += __shfl_down_sync(0xFFFFFFFF, tmp, offset);
        }

        // Write results
        if (threadIdx.x == 0) {
            d_C_reduced[blockIdx.y * n_constraints + constraint_ind] = tmp;
        }
    }
}