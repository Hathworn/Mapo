#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ static float rgbaToGray(uchar4 rgba) {
    return (0.299f * (float)rgba.x + 0.587f * (float)rgba.y +
            0.114f * (float)rgba.z);
}

__global__ void createAnaglyph_kernel(uchar4 *out_image, const uchar4 *left_image, const uchar4 *right_image, int width, int height, int pre_shift) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        uchar4 temp;
        int index = y * width + x; // Compute index once to avoid recomputation
        temp.x = rgbaToGray(left_image[index]);

        // Use ternary operator to simplify condition
        int x_right = x - pre_shift;
        if (x_right > 0 && x_right < width) {
            float gray_value = rgbaToGray(right_image[y * width + x_right]);
            temp.y = gray_value;
            temp.z = gray_value;
        } else {
            temp.y = 0;
            temp.z = 0;
        }

        temp.w = 255; // Alpha value always set to 255

        out_image[index] = temp;
    }
}