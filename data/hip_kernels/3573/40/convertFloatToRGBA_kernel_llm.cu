#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertFloatToRGBA_kernel(uchar4 *out_image, const float *in_image, int width, int height) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int IND = y * width + x;
        float val = in_image[IND];
        uchar4 temp;
        temp.x = temp.y = temp.z = static_cast<unsigned char>(val); // Use a static cast
        temp.w = 255; // Set alpha channel to 255
        out_image[IND] = temp;
    }
}