#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deInterleave_kernel(float *d_X_out, float *d_Y_out, float2 *d_XY_in, int pitch_out, int pitch_in, int width, int height) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure that only valid threads continue
    if (x < width && y < height) {
        // Efficient access using pointers calculation with byte-offsets
        float2 XY = *((float2 *)((char *)d_XY_in + y * pitch_in) + x);
        
        // Use intrinsic functions for memory storage
        float *outX = (float *)((char *)d_X_out + y * pitch_out) + x;
        float *outY = (float *)((char *)d_Y_out + y * pitch_out) + x;
        
        *outX = XY.x;
        *outY = XY.y;
    }
}