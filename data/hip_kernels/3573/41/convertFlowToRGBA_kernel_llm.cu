#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertFlowToRGBA_kernel(uchar4 *d_flowx_out, uchar4 *d_flowy_out, const float *d_flowx_in, const float *d_flowy_in, int width, int height, float lowerLim, float upperLim, float minMag) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return; // Early exit for threads outside the range
    
    uchar4 tempx, tempy;
    float ux = d_flowx_in[y * width + x];
    float uy = d_flowy_in[y * width + x];

    float mag = sqrtf(ux * ux + uy * uy);

    if (!isfinite(ux) || mag < minMag) {
        tempx = {255, 255, 255, 255};
        tempy = {255, 255, 255, 255};
    } else {
        // Rescale value from [lowerLim, upperLim] to [0,1]
        ux = (ux - lowerLim) / (upperLim - lowerLim);
        uy = (uy - lowerLim) / (upperLim - lowerLim);

        // Color calculation for ux
        float r = 1.0f, g = 1.0f, b = 1.0f;
        if (ux < 0.25f) {
            r = 0;
            g = 4.0f * ux;
        } else if (ux < 0.5f) {
            r = 0;
            b = 1.0 + 4.0f * (0.25f - ux);
        } else if (ux < 0.75f) {
            r = 4.0f * (ux - 0.5f);
            b = 0;
        } else {
            g = 1.0f + 4.0f * (0.75f - ux);
            b = 0;
        }
        tempx = {static_cast<unsigned char>(255.0f * r), static_cast<unsigned char>(255.0f * g), static_cast<unsigned char>(255.0f * b), 255};

        // Color calculation for uy
        r = 1.0f;
        g = 1.0f;
        b = 1.0f;
        if (uy < 0.25f) {
            r = 0;
            g = 4.0f * uy;
        } else if (uy < 0.5f) {
            r = 0;
            b = 1.0 + 4.0f * (0.25f - uy);
        } else if (uy < 0.75f) {
            r = 4.0f * (uy - 0.5f);
            b = 0;
        } else {
            g = 1.0f + 4.0f * (0.75f - uy);
            b = 0;
        }
        tempy = {static_cast<unsigned char>(255.0f * r), static_cast<unsigned char>(255.0f * g), static_cast<unsigned char>(255.0f * b), 255};
    }

    d_flowx_out[y * width + x] = tempx;
    d_flowy_out[y * width + x] = tempy;
}