#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "convertFlowToRGBA_kernel_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    uchar4 *d_flowx_out = NULL;
hipMalloc(&d_flowx_out, XSIZE*YSIZE);
uchar4 *d_flowy_out = NULL;
hipMalloc(&d_flowy_out, XSIZE*YSIZE);
const float *d_flowx_in = NULL;
hipMalloc(&d_flowx_in, XSIZE*YSIZE);
const float *d_flowy_in = NULL;
hipMalloc(&d_flowy_in, XSIZE*YSIZE);
int width = XSIZE;
int height = YSIZE;
float lowerLim = 1;
float upperLim = 1;
float minMag = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
convertFlowToRGBA_kernel<<<gridBlock, threadBlock>>>(d_flowx_out,d_flowy_out,d_flowx_in,d_flowy_in,width,height,lowerLim,upperLim,minMag);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
convertFlowToRGBA_kernel<<<gridBlock, threadBlock>>>(d_flowx_out,d_flowy_out,d_flowx_in,d_flowy_in,width,height,lowerLim,upperLim,minMag);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
convertFlowToRGBA_kernel<<<gridBlock, threadBlock>>>(d_flowx_out,d_flowy_out,d_flowx_in,d_flowy_in,width,height,lowerLim,upperLim,minMag);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}