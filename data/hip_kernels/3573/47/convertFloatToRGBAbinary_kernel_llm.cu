#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertFloatToRGBAbinary_kernel(uchar4 *out_image, const float *in_image, int width, int height, float lowerLim, float upperLim) {
    // Calculate global index using block and thread indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Proceed only if (x, y) is within image bounds
    if (x < width && y < height) {
        // Simplified index calculation
        int idx = y * width + x;
        float val = in_image[idx];

        uchar4 temp;
        // Set pixel to white if invalid or out of specified limits, otherwise black
        if (!isfinite(val) || val < lowerLim || val > upperLim) {
            temp = {255, 255, 255, 255};  // White
        } else {
            temp = {0, 0, 0, 0};          // Black
        }
        out_image[idx] = temp;
    }
}