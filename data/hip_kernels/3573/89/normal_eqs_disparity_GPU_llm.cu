#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normal_eqs_disparity_GPU(float *d_CD, const float *d_disparity_compact, const float4 *d_Zbuffer_normals_compact, const int *d_ind_disparity_Zbuffer, float fx, float fy, float ox, float oy, float b, int n_cols, const int *d_n_values_disparity, const int *d_start_ind_disparity, float w_disp) {
    int n_val_accum = gridDim.x * blockDim.x;
    int n_disparity = d_n_values_disparity[blockIdx.y];
    int n_accum = (int)ceilf((float)n_disparity / (float)n_val_accum);
    int start_ind = d_start_ind_disparity[blockIdx.y];

    // initialize accumulators
    float accumulators[27] = {0.0f};

    for (int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x;
         in_ind < blockDim.x * (blockIdx.x + 1) * n_accum; in_ind += blockDim.x) {

        if (in_ind < n_disparity) {
            // fetch disparity, Zbuffer and normal from global memory
            float disp = d_disparity_compact[in_ind + start_ind];
            float4 tmp = d_Zbuffer_normals_compact[in_ind + start_ind];
            float Zbuffer = tmp.x;
            float nx = tmp.y;
            float ny = tmp.z;
            float nz = tmp.w;

            // compute coordinates
            int pixel_ind = d_ind_disparity_Zbuffer[in_ind + start_ind];
            float y = floorf(__fdividef((float)pixel_ind, n_cols));
            float x = (float)pixel_ind - y * n_cols;
            x = __fdividef((x - ox), fx);
            y = __fdividef((y - oy), fy);

            // reconstruct 3D point from disparity
            float Zd = -(fx * b) / disp;
            float Xd = x * Zd;
            float Yd = y * Zd;

            // reconstruct 3D point from model
            float Zm = Zbuffer;
            float Xm = x * Zm;
            float Ym = y * Zm;

            // weight the constraint conversion
            float w2 = fx * b / (Zm * Zm);
            w2 *= w2;

            // evaluate constraints
            accumulators[0] += w2 * (nx * nx);
            accumulators[1] += w2 * (nx * ny);
            accumulators[2] += w2 * (nx * nz);
            accumulators[3] += w2 * (Ym * nx * nz - Zm * nx * ny);
            accumulators[4] += w2 * (Zm * (nx * nx) - Xm * nx * nz);
            accumulators[5] += w2 * (-Ym * (nx * nx) + Xm * nx * ny);

            accumulators[6] += w2 * (ny * ny);
            accumulators[7] += w2 * (ny * nz);
            accumulators[8] += w2 * (-Zm * (ny * ny) + Ym * ny * nz);
            accumulators[9] += w2 * (-Xm * ny * nz + Zm * nx * ny);
            accumulators[10] += w2 * (Xm * (ny * ny) - Ym * nx * ny);

            accumulators[11] += w2 * (nz * nz);
            accumulators[12] += w2 * (Ym * (nz * nz) - Zm * ny * nz);
            accumulators[13] += w2 * (-Xm * (nz * nz) + Zm * nx * nz);
            accumulators[14] += w2 * (Xm * ny * nz - Ym * nx * nz);

            accumulators[15] += w2 * ((Ym * Ym) * (nz * nz) + (Zm * Zm) * (ny * ny) -
                                      Ym * Zm * ny * nz * 2.0f);
            accumulators[16] += w2 * (-Xm * Ym * (nz * nz) - (Zm * Zm) * nx * ny +
                                      Xm * Zm * ny * nz + Ym * Zm * nx * nz);
            accumulators[17] += w2 * (-Xm * Zm * (ny * ny) - (Ym * Ym) * nx * nz +
                                      Xm * Ym * ny * nz + Ym * Zm * nx * ny);

            accumulators[18] += w2 * ((Xm * Xm) * (nz * nz) + (Zm * Zm) * (nx * nx) -
                                      Xm * Zm * nx * nz * 2.0f);
            accumulators[19] += w2 * (-Ym * Zm * (nx * nx) - (Xm * Xm) * ny * nz +
                                      Xm * Ym * nx * nz + Xm * Zm * nx * ny);

            accumulators[20] += w2 * ((Xm * Xm) * (ny * ny) + (Ym * Ym) * (nx * nx) -
                                      Xm * Ym * nx * ny * 2.0f);

            // B-vector
            accumulators[21] += w2 * (Xd * (nx * nx) - Xm * (nx * nx) + Yd * nx * ny -
                                      Ym * nx * ny + Zd * nx * nz - Zm * nx * nz);
            accumulators[22] += w2 * (Yd * (ny * ny) - Ym * (ny * ny) + Xd * nx * ny -
                                      Xm * nx * ny + Zd * ny * nz - Zm * ny * nz);
            accumulators[23] += w2 * (Zd * (nz * nz) - Zm * (nz * nz) + Xd * nx * nz -
                                      Xm * nx * nz + Yd * ny * nz - Ym * ny * nz);
            accumulators[24] += w2 *
                                (-Yd * Zm * (ny * ny) + Ym * Zd * (nz * nz) + Ym * Zm * (ny * ny) -
                                 Ym * Zm * (nz * nz) - (Ym * Ym) * ny * nz + (Zm * Zm) * ny * nz +
                                 Xd * Ym * nx * nz - Xm * Ym * nx * nz - Xd * Zm * nx * ny +
                                 Yd * Ym * ny * nz + Xm * Zm * nx * ny - Zd * Zm * ny * nz);
            accumulators[25] += w2 *
                                (Xd * Zm * (nx * nx) - Xm * Zd * (nz * nz) - Xm * Zm * (nx * nx) +
                                 Xm * Zm * (nz * nz) + (Xm * Xm) * nx * nz - (Zm * Zm) * nx * nz -
                                 Xd * Xm * nx * nz - Xm * Yd * ny * nz + Xm * Ym * ny * nz +
                                 Yd * Zm * nx * ny - Ym * Zm * nx * ny + Zd * Zm * nx * nz);
            accumulators[26] += w2 *
                                (-Xd * Ym * (nx * nx) + Xm * Yd * (ny * ny) + Xm * Ym * (nx * nx) -
                                 Xm * Ym * (ny * ny) - (Xm * Xm) * nx * ny + (Ym * Ym) * nx * ny +
                                 Xd * Xm * nx * ny - Yd * Ym * nx * ny + Xm * Zd * ny * nz -
                                 Xm * Zm * ny * nz - Ym * Zd * nx * nz + Ym * Zm * nx * nz);
        }
    }

    // write out accumulators
    int out_ind = 27 * n_val_accum * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    w_disp *= w_disp;

    for (int i = 0; i < 27; ++i) {
        d_CD[out_ind + i * n_val_accum] = w_disp * accumulators[i];
    }
}