#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertKinectDisparityInPlace_kernel(float *d_disparity, int pitch, int width, int height, float depth_scale) {
    // Calculate the 2D position in the grid
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within image bounds
    if (x < width && y < height) {
        // Calculate the input pointer for the current position
        float *d_in = reinterpret_cast<float*>(reinterpret_cast<char*>(d_disparity) + y * pitch) + x;
        
        // Update disparity value or set to NaN
        *d_in = (*d_in == 0.0f) ? __int_as_float(0x7fffffff) : (-depth_scale / *d_in);  // Faster use of integer representation of NaN
    }
}