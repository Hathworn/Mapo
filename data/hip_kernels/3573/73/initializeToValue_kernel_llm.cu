#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initializeToValue_kernel(unsigned int *data, unsigned int value, int width, int height) {
    // Calculate 1D index based on 2D grid dimensions
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int offset = gridDim.x * blockDim.x; // Total number of threads in a row

    // Iterate over all elements row by row
    for (int i = idx; i < width * height; i += offset) {
        data[i] = value;
    }
}