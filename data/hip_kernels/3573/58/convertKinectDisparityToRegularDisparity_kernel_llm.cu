#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertKinectDisparityToRegularDisparity_kernel(float *d_regularDisparity, int d_regularDisparityPitch, 
                                                                const float *d_KinectDisparity, int d_KinectDisparityPitch, 
                                                                int width, int height) {

    // Calculate global thread indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread is within image boundaries
    if (x < width && y < height) {
        // Efficiently calculate input and output pointers
        const float *inputPtr = (const float*)((const char*)d_KinectDisparity + y * d_KinectDisparityPitch) + x;
        float *outputPtr = (float*)((char*)d_regularDisparity + y * d_regularDisparityPitch) + x;

        // Read input and compute output value
        float d_in = *inputPtr;
        float d_out = (d_in == 0.0f) ? nanf("") : -d_in;

        // Write computed output
        *outputPtr = d_out;
    }
}