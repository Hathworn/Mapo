#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ static float flow_absolute_residual(float x, float y, float ux, float uy, float d, float fx, float fy, float T0, float T1, float T2, float R0, float R1, float R2) {
    float rx = -ux + fx * R1 - y * R2 + ((x * x) * R1) / fx + d * fx * T0 - d * x * T2 - (x * y * R0) / fx;
    float ry = -uy - fy * R0 + x * R2 - d * y * T2 - ((y * y) * R0) / fy + d * fy * T1 + (x * y * R1) / fy;
    return sqrtf(rx * rx + ry * ry);
}

__global__ void normal_eqs_flow_weighted_GPU(float *d_CO, const float2 *d_flow_compact, const float *d_Zbuffer_flow_compact, const int *d_ind_flow_Zbuffer, float fx, float fy, float ox, float oy, int n_rows, int n_cols, const int *d_n_values_flow, const int *d_start_ind_flow, const float *d_abs_res_scales, float w_flow, float w_ar_flow, const float *d_dTR) {

    int n_val_accum = gridDim.x * blockDim.x;
    int n_flow = d_n_values_flow[blockIdx.y];
    int n_accum = (int)ceilf((float)n_flow / (float)n_val_accum);
    int start_ind = d_start_ind_flow[blockIdx.y];

    // Use shared memory for accumulation to reduce global memory transactions
    __shared__ float shared_A[23];
    for(int i = threadIdx.x; i < 23; i += blockDim.x) {
        shared_A[i] = 0.0f;
    }
    __syncthreads();

    for (int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x; in_ind < blockDim.x * (blockIdx.x + 1) * n_accum; in_ind += blockDim.x) {
        if (in_ind < n_flow) {
            // Fetch flow and Zbuffer from global memory
            float2 u = d_flow_compact[in_ind + start_ind];
            float disp = __fdividef(1.0f, d_Zbuffer_flow_compact[in_ind + start_ind]);

            // Compute coordinates
            int pixel_ind = d_ind_flow_Zbuffer[in_ind + start_ind];
            bool is_ar_flow = (pixel_ind >= (n_rows * n_cols));
            pixel_ind -= (int)is_ar_flow * n_rows * n_cols;

            float y = floorf(__fdividef((float)pixel_ind, n_cols));
            float x = (float)pixel_ind - y * n_cols;

            x = x - ox;
            y = y - oy;

            // Determine M-estimation weight
            float w_rel = is_ar_flow ? w_ar_flow : w_flow;
            int s6 = blockIdx.y * 6;
            float w = w_rel * flow_absolute_residual(x, y, u.x, u.y, disp, fx, fy, d_dTR[s6], d_dTR[s6 + 1], d_dTR[s6 + 2], d_dTR[s6 + 3], d_dTR[s6 + 4], d_dTR[s6 + 5]);
            w /= d_abs_res_scales[blockIdx.y];
            w = (w > 1) ? 0 : (1.0f - 2.0f * w * w + w * w * w * w);

            // Evaluate constraints and update shared memory
            atomicAdd(&shared_A[0], w * (disp * disp * fx * fx));
            atomicAdd(&shared_A[1], w * (-disp * disp * x * fx));
            atomicAdd(&shared_A[2], w * (-disp * x * y));
            atomicAdd(&shared_A[3], w * (disp * fx * fx + disp * x * x));
            atomicAdd(&shared_A[4], w * (-disp * y * fx));
            atomicAdd(&shared_A[5], w * (-disp * disp * y * fy));
            atomicAdd(&shared_A[6], w * (-disp * fy * fy - disp * y * y));
            atomicAdd(&shared_A[7], w * (disp * x * fy));
            atomicAdd(&shared_A[8], w * (disp * disp * x * x + disp * disp * y * y));
            atomicAdd(&shared_A[9], w * (disp * x * x * y / fx + disp * y * fy + disp * y * y * y / fy));
            atomicAdd(&shared_A[10], w * (-disp * x * fx - disp * x * x * x / fx - disp * x * y * y / fy));
            atomicAdd(&shared_A[11], w * (x * x * y * y / (fx * fx) + fy * fy + 2.0f * y * y + y * y * y * y / (fy * fy)));
            atomicAdd(&shared_A[12], w * (-2.0f * x * y - x * x * x * y / (fx * fx) - x * y * y * y / (fy * fy)));
            atomicAdd(&shared_A[13], w * (x * y * y / fx - x * fy - x * y * y / fy));
            atomicAdd(&shared_A[14], w * (fx * fx + 2.0f * x * x + x * x * x * x / (fx * fx) + x * x * y * y / (fy * fy)));
            atomicAdd(&shared_A[15], w * (-y * fx - x * x * y / fx + x * x * y / fy));
            atomicAdd(&shared_A[16], w * (x * x + y * y));

            // B-vector
            atomicAdd(&shared_A[17], w * (disp * u.x * fx));
            atomicAdd(&shared_A[18], w * (disp * u.y * fy));
            atomicAdd(&shared_A[19], w * (-disp * x * u.x - disp * y * u.y));
            atomicAdd(&shared_A[20], w * (-x * y * u.x / fx - u.y * fy - u.y * y * y / fy));
            atomicAdd(&shared_A[21], w * (u.x * fx + x * x * u.x / fx + x * y * u.y / fy));
            atomicAdd(&shared_A[22], w * (-y * u.x + x * u.y));
        }
    }

    __syncthreads();

    // Write out accumulators from shared to global memory
    int out_ind = 23 * n_val_accum * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    for(int i = threadIdx.x; i < 23; i += blockDim.x) {
        d_CO[out_ind + i * n_val_accum] = shared_A[i];
    }
}