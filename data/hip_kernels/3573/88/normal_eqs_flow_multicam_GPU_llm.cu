#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normal_eqs_flow_multicam_GPU(float *d_CO, float2 *d_flow_compact, float *d_Zbuffer_flow_compact, int *d_ind_flow_Zbuffer, const float *d_focal_length, const float *d_nodal_point_x, const float *d_nodal_point_y, const int *d_n_rows, const int *d_n_cols, const int *d_n_values_flow, const int *d_start_ind_flow, const int *d_pixel_ind_offset) {
    // Shared memory for partial accumulators
    __shared__ float shared_A[23 * BLOCK_SIZE];

    // Thread indexes
    int threadID = threadIdx.x;
    
    // Shared memory initialization
    if (threadID < 23) {
        shared_A[threadID * BLOCK_SIZE + threadID] = 0.0f;
    }
    __syncthreads();

    int n_val_accum = gridDim.x * blockDim.x;
    int n_flow = d_n_values_flow[blockIdx.y];
    int n_accum = (int)ceilf((float)n_flow / (float)n_val_accum);
    int start_ind = d_start_ind_flow[blockIdx.y];

    float f = d_focal_length[blockIdx.y];
    float ox = d_nodal_point_x[blockIdx.y];
    float oy = d_nodal_point_y[blockIdx.y];
    int n_rows = d_n_rows[blockIdx.y];
    int n_cols = d_n_cols[blockIdx.y];
    int pixel_ind_offset = d_pixel_ind_offset[blockIdx.y];

    for (int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x; in_ind < blockDim.x * (blockIdx.x + 1) * n_accum; in_ind += blockDim.x) {
        if (in_ind < n_flow) {
            float2 u = d_flow_compact[in_ind + start_ind];
            float disp = __fdividef(1.0f, d_Zbuffer_flow_compact[in_ind + start_ind]);
            int pixel_ind = d_ind_flow_Zbuffer[in_ind + start_ind] - pixel_ind_offset;
            bool is_ar_flow = (pixel_ind >= (n_rows * n_cols));
            pixel_ind -= (int)is_ar_flow * n_rows * n_cols;

            float y = floorf(__fdividef((float)pixel_ind, n_cols));
            float x = (float)pixel_ind - y * n_cols;

            x = x - ox;
            y = y - oy;

            y = -y;
            u.y = -u.y;

            shared_A[threadID] += (disp * disp * f * f);
            shared_A[BLOCK_SIZE + threadID] += (-disp * disp * x * f);
            shared_A[2 * BLOCK_SIZE + threadID] += (-disp * x * y);
            shared_A[3 * BLOCK_SIZE + threadID] += (disp * f * f + disp * x * x);
            shared_A[4 * BLOCK_SIZE + threadID] += (-disp * y * f);
            shared_A[5 * BLOCK_SIZE + threadID] += (-disp * disp * y * f);
            shared_A[6 * BLOCK_SIZE + threadID] += (-disp * f * f - disp * y * y);
            shared_A[7 * BLOCK_SIZE + threadID] += (disp * x * f);
            shared_A[8 * BLOCK_SIZE + threadID] += (disp * disp * x * x + disp * disp * y * y);
            shared_A[9 * BLOCK_SIZE + threadID] += (disp * x * x * y / f + disp * y * f + disp * y * y * y / f);
            shared_A[10 * BLOCK_SIZE + threadID] += (-disp * x * f - disp * x * x * x / f - disp * x * y * y / f);
            shared_A[11 * BLOCK_SIZE + threadID] += (x * x * y * y / (f * f) + f * f + 2.0f * y * y + y * y * y * y / (f * f));
            shared_A[12 * BLOCK_SIZE + threadID] += (-2.0f * x * y - x * x * x * y / (f * f) - x * y * y * y / (f * f));
            shared_A[13 * BLOCK_SIZE + threadID] += (-x * f);
            shared_A[14 * BLOCK_SIZE + threadID] += (f * f + 2.0f * x * x + x * x * x * x / (f * f) + x * x * y * y / (f * f));
            shared_A[15 * BLOCK_SIZE + threadID] += (-y * f);
            shared_A[16 * BLOCK_SIZE + threadID] += (x * x + y * y);

            shared_A[17 * BLOCK_SIZE + threadID] += (disp * u.x * f);
            shared_A[18 * BLOCK_SIZE + threadID] += (disp * u.y * f);
            shared_A[19 * BLOCK_SIZE + threadID] += (-disp * x * u.x - disp * y * u.y);
            shared_A[20 * BLOCK_SIZE + threadID] += (-x * y * u.x / f - u.y * f - u.y * y * y / f);
            shared_A[21 * BLOCK_SIZE + threadID] += (u.x * f + x * x * u.x / f + x * y * u.y / f);
            shared_A[22 * BLOCK_SIZE + threadID] += (-y * u.x + x * u.y);
        }
    }

    __syncthreads();

    int out_ind = 23 * n_val_accum * blockIdx.y + blockDim.x * blockIdx.x + threadID;

    if (threadID < 23) {
        d_CO[out_ind] = shared_A[threadID * BLOCK_SIZE];
        d_CO[out_ind + n_val_accum] = shared_A[threadID * BLOCK_SIZE + BLOCK_SIZE];
        d_CO[out_ind + 2 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 2 * BLOCK_SIZE];
        d_CO[out_ind + 3 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 3 * BLOCK_SIZE];
        d_CO[out_ind + 4 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 4 * BLOCK_SIZE];
        d_CO[out_ind + 5 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 5 * BLOCK_SIZE];
        d_CO[out_ind + 6 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 6 * BLOCK_SIZE];
        d_CO[out_ind + 7 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 7 * BLOCK_SIZE];
        d_CO[out_ind + 8 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 8 * BLOCK_SIZE];
        d_CO[out_ind + 9 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 9 * BLOCK_SIZE];
        d_CO[out_ind + 10 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 10 * BLOCK_SIZE];
        d_CO[out_ind + 11 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 11 * BLOCK_SIZE];
        d_CO[out_ind + 12 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 12 * BLOCK_SIZE];
        d_CO[out_ind + 13 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 13 * BLOCK_SIZE];
        d_CO[out_ind + 14 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 14 * BLOCK_SIZE];
        d_CO[out_ind + 15 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 15 * BLOCK_SIZE];
        d_CO[out_ind + 16 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 16 * BLOCK_SIZE];
        d_CO[out_ind + 17 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 17 * BLOCK_SIZE];
        d_CO[out_ind + 18 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 18 * BLOCK_SIZE];
        d_CO[out_ind + 19 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 19 * BLOCK_SIZE];
        d_CO[out_ind + 20 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 20 * BLOCK_SIZE];
        d_CO[out_ind + 21 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 21 * BLOCK_SIZE];
        d_CO[out_ind + 22 * n_val_accum] = shared_A[threadID * BLOCK_SIZE + 22 * BLOCK_SIZE];
    }
}
```
