#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertPitchedFloatToGrayRGBA_kernel(uchar4 *out_image, const float *in_image, int width, int height, int pitch, float lowerLim, float upperLim) {
    // Calculate unique thread indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        // Optimize memory access by using pitched pointer arithmetic
        float val = *((float *)((char *)in_image + y * pitch) + x);

        // Rescale value to [0,255]
        val = (val - lowerLim) / (upperLim - lowerLim) * 255.0f;

        // Assign result to output image
        uchar4 temp;
        temp.x = temp.y = temp.z = static_cast<unsigned char>(val);
        temp.w = 255;
        out_image[y * width + x] = temp;
    }
}