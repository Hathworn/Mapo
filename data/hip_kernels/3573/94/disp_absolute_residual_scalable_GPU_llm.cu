#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ static float disp_absolute_residual(float Xd, float Yd, float Zd, float Xm, float Ym, float Zm, float nx, float ny, float nz, float T0, float T1, float T2, float R0, float R1, float R2, float fx, float b) {
    float r = -Xd * nx + Xm * nx - Yd * ny + Ym * ny - Zd * nz + Zm * nz +
              nx * T0 + ny * T1 + nz * T2 + Xm * ny * R2 - Xm * nz * R1 -
              Ym * nx * R2 + Ym * nz * R0 + Zm * nx * R1 - Zm * ny * R0;

    // Weight to convert distance units to pixels
    r *= fx * b / (Zm * Zm);

    return fabsf(r);
}

__global__ void disp_absolute_residual_scalable_GPU(float *d_abs_res, const float *d_disparity_compact, const float4 *d_Zbuffer_normals_compact, const int *d_ind_disparity_Zbuffer, const unsigned int *d_valid_disparity_Zbuffer, float fx, float fy, float ox, float oy, float b, int n_cols, int n_valid_disparity_Zbuffer, const int *d_offset_ind, const int *d_segment_translation_table, float w_disp, const float *d_dTR) {

    // Use shared memory to reduce global memory accesses
    extern __shared__ float shmem[];
    float *sh_dTR = shmem;

    int ind = blockDim.x * blockIdx.x + threadIdx.x;

    // Load transformation matrix into shared memory
    if (threadIdx.x < 6) {
        sh_dTR[threadIdx.x] = d_dTR[threadIdx.x + blockIdx.x * 6];
    }
    __syncthreads();

    // Only process valid indices
    if (ind < n_valid_disparity_Zbuffer) {

        // Determine current segment
        int segment = d_segment_translation_table[d_valid_disparity_Zbuffer[ind]];

        // Fetch disparity, Zbuffer, and normal from global memory
        float disp = d_disparity_compact[ind];
        float4 tmp = d_Zbuffer_normals_compact[ind];
        float Zbuffer = tmp.x;
        float nx = tmp.y;
        float ny = tmp.z;
        float nz = tmp.w;

        // Compute coordinates
        int pixel_ind = d_ind_disparity_Zbuffer[ind];
        float y = floorf((float)pixel_ind / n_cols);
        float x = (float)pixel_ind - y * n_cols;

        x = (x - ox) / fx;
        y = (y - oy) / fy;

        // Reconstruct 3D point from disparity
        float Zd = -(fx * b) / disp;  // Arbitrary use of fx for now
        float Xd = x * Zd;
        float Yd = y * Zd;

        // Reconstruct 3D point from model
        float Zm = Zbuffer;
        float Xm = x * Zm;
        float Ym = y * Zm;

        // Compute absolute residual (weighted by disparity vs flow importance)
        int ind_out = ind + d_offset_ind[segment];
        d_abs_res[ind_out] = w_disp * disp_absolute_residual(Xd, Yd, Zd, Xm, Ym, Zm, nx, ny, nz,
                                                             sh_dTR[0], sh_dTR[1], sh_dTR[2],
                                                             sh_dTR[3], sh_dTR[4], sh_dTR[5],
                                                             fx, b);
    }
}