#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void invalidateFlow_kernel(float *modFlowX, float *modFlowY, const float *constFlowX, const float *constFlowY, int width, int height, float cons_thres) {
    // Compute global thread index for the 2D grid
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within bounds
    if (x < width && y < height) {
        int ind = y * width + x;  // Calculate 1D index

        // Load flow values
        float mFX = modFlowX[ind];
        float mFY = modFlowY[ind];
        float cFX = constFlowX[ind];
        float cFY = constFlowY[ind];

        // Calculate error
        float err = (mFX - cFX) * (mFX - cFX) + (mFY - cFY) * (mFY - cFY);
        err = sqrtf(err);

        // Invalidate flow if error is above threshold
        if (err > cons_thres) {
            modFlowX[ind] = nanf("");
            modFlowY[ind] = nanf("");
        } else {
            modFlowX[ind] = mFX;
            modFlowY[ind] = mFY;
        }
    }
}