#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertPointCloudToDepthImage_kernel(unsigned int *depth_image, const float4 *point_cloud, int n_cols, int n_rows, int n_points, float nodal_point_x, float nodal_point_y, float focal_length_x, float focal_length_y, const float *T, const float *R) {
    const int ind = blockIdx.x * blockDim.x + threadIdx.x;

    if (ind < n_points) {
        // Fetch point
        float4 point = point_cloud[ind];

        // Transform to camera coordinates
        float x = R[0] * point.x + R[1] * point.y + R[2] * point.z + T[0];
        float y = R[3] * point.x + R[4] * point.y + R[5] * point.z + T[1];
        float z = R[6] * point.x + R[7] * point.y + R[8] * point.z + T[2];

        float inv_z = __frcp_rn(z); // Fast reciprocal

        // Project to image coordinates
        int x_pix = __float2int_rn(focal_length_x * x * inv_z + nodal_point_x);
        int y_pix = __float2int_rn(focal_length_y * y * inv_z + nodal_point_y);

        // Check and update if point is valid
        if (x_pix >= 0 && x_pix < n_cols && y_pix >= 0 && y_pix < n_rows) {
            int ind_out = y_pix * n_cols + x_pix;
            atomicMin(&depth_image[ind_out], (unsigned int)(point.z * 1000.0f)); // Use atomicMin for proper concurrency
        }
    }
}