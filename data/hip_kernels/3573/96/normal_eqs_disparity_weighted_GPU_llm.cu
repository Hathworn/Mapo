#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ static float disp_absolute_residual(float Xd, float Yd, float Zd, float Xm, float Ym, float Zm, float nx, float ny, float nz, float T0, float T1, float T2, float R0, float R1, float R2, float fx, float b) {
    float r = -Xd * nx + Xm * nx - Yd * ny + Ym * ny - Zd * nz + Zm * nz +
              nx * T0 + ny * T1 + nz * T2 + Xm * ny * R2 - Xm * nz * R1 -
              Ym * nx * R2 + Ym * nz * R0 + Zm * nx * R1 - Zm * ny * R0;

    // weight to convert distance units to pixels
    r *= fx * b / (Zm * Zm);

    return fabsf(r);
}

__global__ void normal_eqs_disparity_weighted_GPU(float *d_CD, const float *d_disparity_compact, const float4 *d_Zbuffer_normals_compact, const int *d_ind_disparity_Zbuffer, float fx, float fy, float ox, float oy, float b, int n_cols, const int *d_n_values_disparity, const int *d_start_ind_disparity, const float *d_abs_res_scales, float w_disp, const float *d_dTR) {
    // Utilize shared memory for accumulation
    extern __shared__ float shared_A[27];
    for (int i = 0; i < 27; ++i) shared_A[i] = 0.0f;

    int n_val_accum = gridDim.x * blockDim.x;
    int n_disparity = d_n_values_disparity[blockIdx.y];
    int n_accum = (int)ceilf((float)n_disparity / (float)n_val_accum);
    int start_ind = d_start_ind_disparity[blockIdx.y];

    for (int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x; in_ind < blockDim.x * (blockIdx.x + 1) * n_accum; in_ind += blockDim.x) {
        if (in_ind < n_disparity) {
            // fetch disparity, Zbuffer and normal from global memory
            float disp = d_disparity_compact[in_ind + start_ind];
            float4 tmp = d_Zbuffer_normals_compact[in_ind + start_ind];
            float Zbuffer = tmp.x;
            float nx = tmp.y;
            float ny = tmp.z;
            float nz = tmp.w;

            // compute coordinates
            int pixel_ind = d_ind_disparity_Zbuffer[in_ind + start_ind];
            float y = floorf(__fdividef((float)pixel_ind, n_cols));
            float x = (float)pixel_ind - y * n_cols;
            x = __fdividef((x - ox), fx);
            y = __fdividef((y - oy), fy);

            // reconstruct 3D point from disparity
            float Zd = -(fx * b) / disp; // arbitrary use of fx
            float Xd = x * Zd;
            float Yd = y * Zd;

            // reconstruct 3D point from model
            float Zm = Zbuffer;
            float Xm = x * Zm;
            float Ym = y * Zm;

            // determine M-estimation weight
            // disparity residual weighed by rel. importance disp vs flow
            int s6 = blockIdx.y * 6;
            float w = w_disp * disp_absolute_residual(Xd, Yd, Zd, Xm, Ym, Zm, nx, ny, nz, d_dTR[s6], d_dTR[s6 + 1], d_dTR[s6 + 2], d_dTR[s6 + 3], d_dTR[s6 + 4], d_dTR[s6 + 5], fx, b);
            w /= d_abs_res_scales[blockIdx.y];
            w = (w > 1) ? 0 : (1.0f - 2.0f * w * w + w * w * w * w);

            // multiply m estimation weight with distance->pixel conversion weight (squared)
            w *= (fx * fx * b * b) / (Zm * Zm * Zm * Zm);

            /************************/
            /* evaluate constraints */
            /************************/

            // unique values A-matrix
            atomicAdd(&shared_A[0], w * (nx * nx));
            atomicAdd(&shared_A[1], w * (nx * ny));
            atomicAdd(&shared_A[2], w * (nx * nz));
            atomicAdd(&shared_A[3], w * (Ym * nx * nz - Zm * nx * ny));
            atomicAdd(&shared_A[4], w * (Zm * (nx * nx) - Xm * nx * nz));
            atomicAdd(&shared_A[5], w * (-Ym * (nx * nx) + Xm * nx * ny));

            atomicAdd(&shared_A[6], w * (ny * ny));
            atomicAdd(&shared_A[7], w * (ny * nz));
            atomicAdd(&shared_A[8], w * (-Zm * (ny * ny) + Ym * ny * nz));
            atomicAdd(&shared_A[9], w * (-Xm * ny * nz + Zm * nx * ny));
            atomicAdd(&shared_A[10], w * (Xm * (ny * ny) - Ym * nx * ny));

            atomicAdd(&shared_A[11], w * (nz * nz));
            atomicAdd(&shared_A[12], w * (Ym * (nz * nz) - Zm * ny * nz));
            atomicAdd(&shared_A[13], w * (-Xm * (nz * nz) + Zm * nx * nz));
            atomicAdd(&shared_A[14], w * (Xm * ny * nz - Ym * nx * nz));

            atomicAdd(&shared_A[15], w * ((Ym * Ym) * (nz * nz) + (Zm * Zm) * (ny * ny) - Ym * Zm * ny * nz * 2.0f));
            atomicAdd(&shared_A[16], w * (-Xm * Ym * (nz * nz) - (Zm * Zm) * nx * ny + Xm * Zm * ny * nz + Ym * Zm * nx * nz));
            atomicAdd(&shared_A[17], w * (-Xm * Zm * (ny * ny) - (Ym * Ym) * nx * nz + Xm * Ym * ny * nz + Ym * Zm * nx * ny));

            atomicAdd(&shared_A[18], w * ((Xm * Xm) * (nz * nz) + (Zm * Zm) * (nx * nx) - Xm * Zm * nx * nz * 2.0f));
            atomicAdd(&shared_A[19], w * (-Ym * Zm * (nx * nx) - (Xm * Xm) * ny * nz + Xm * Ym * nx * nz + Xm * Zm * nx * ny));

            atomicAdd(&shared_A[20], w * ((Xm * Xm) * (ny * ny) + (Ym * Ym) * (nx * nx) - Xm * Ym * nx * ny * 2.0f));

            // B-vector
            atomicAdd(&shared_A[21], w * (Xd * (nx * nx) - Xm * (nx * nx) + Yd * nx * ny - Ym * nx * ny + Zd * nx * nz - Zm * nx * nz));
            atomicAdd(&shared_A[22], w * (Yd * (ny * ny) - Ym * (ny * ny) + Xd * nx * ny - Xm * nx * ny + Zd * ny * nz - Zm * ny * nz));
            atomicAdd(&shared_A[23], w * (Zd * (nz * nz) - Zm * (nz * nz) + Xd * nx * nz - Xm * nx * nz + Yd * ny * nz - Ym * ny * nz));
            atomicAdd(&shared_A[24], w * (-Yd * Zm * (ny * ny) + Ym * Zd * (nz * nz) + Ym * Zm * (ny * ny) - Ym * Zm * (nz * nz) - (Ym * Ym) * ny * nz + (Zm * Zm) * ny * nz + Xd * Ym * nx * nz - Xm * Ym * nx * nz - Xd * Zm * nx * ny + Yd * Ym * ny * nz + Xm * Zm * nx * ny - Zd * Zm * ny * nz));
            atomicAdd(&shared_A[25], w * (Xd * Zm * (nx * nx) - Xm * Zd * (nz * nz) - Xm * Zm * (nx * nx) + Xm * Zm * (nz * nz) + (Xm * Xm) * nx * nz - (Zm * Zm) * nx * nz - Xd * Xm * nx * nz - Xm * Yd * ny * nz + Xm * Ym * ny * nz + Yd * Zm * nx * ny - Ym * Zm * nx * ny + Zd * Zm * nx * nz));
            atomicAdd(&shared_A[26], w * (-Xd * Ym * (nx * nx) + Xm * Yd * (ny * ny) + Xm * Ym * (nx * nx) - Xm * Ym * (ny * ny) - (Xm * Xm) * nx * ny + (Ym * Ym) * nx * ny + Xd * Xm * nx * ny - Yd * Ym * nx * ny + Xm * Zd * ny * nz - Xm * Zm * ny * nz - Ym * Zd * nx * nz + Ym * Zm * nx * nz));
        }
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        int out_ind = 27 * n_val_accum * blockIdx.y + blockDim.x * blockIdx.x;
        w_disp *= w_disp; // weight relative to flow
        for (int i = 0; i < 27; ++i) {
            d_CD[out_ind + i * n_val_accum] = w_disp * shared_A[i];
        }
    }
}