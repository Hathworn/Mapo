#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void run_reduction(int *con, int *blockCon, int *ActiveList, int nActiveBlock, int *blockSizes)
{
    int list_idx = blockIdx.x;
    int tx = threadIdx.x;
    int block_idx = ActiveList[list_idx];
    int start = block_idx * blockDim.x * 2;
    int blocksize = blockSizes[block_idx];
    __shared__ int s_block_conv;
    
    // Initialize shared memory once for all threads in the block
    if (tx == 0) {
        s_block_conv = 1;
    }
    __syncthreads();
    
    // Use stride loop to allow all threads to check conditions
    for (int i = tx; i < blocksize; i += blockDim.x) {
        if (!con[start + i]) {
            atomicExch(&s_block_conv, 0);
        }
    }
    __syncthreads();

    // Write final result by the first thread
    if (tx == 0) {
        blockCon[block_idx] = s_block_conv; // active list is negation of tile convergence (active = not converged)
    }
}