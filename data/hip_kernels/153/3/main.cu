#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "CopyOutBack.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float4 *d_tetT = NULL;
hipMalloc(&d_tetT, XSIZE*YSIZE);
float *d_vertT = NULL;
hipMalloc(&d_vertT, XSIZE*YSIZE);
int *d_vertMem = NULL;
hipMalloc(&d_vertMem, XSIZE*YSIZE);
int *d_vertMemOutside = NULL;
hipMalloc(&d_vertMemOutside, XSIZE*YSIZE);
int *d_BlockSizes = NULL;
hipMalloc(&d_BlockSizes, XSIZE*YSIZE);
int *d_ActiveList = NULL;
hipMalloc(&d_ActiveList, XSIZE*YSIZE);
int m_maxNumInVert = 1;
int m_maxNumTotalTets = 1;
int m_maxVertMappingInside = 1;
int m_maxVertMappingOutside = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
CopyOutBack<<<gridBlock, threadBlock>>>(d_tetT,d_vertT,d_vertMem,d_vertMemOutside,d_BlockSizes,d_ActiveList,m_maxNumInVert,m_maxNumTotalTets,m_maxVertMappingInside,m_maxVertMappingOutside);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
CopyOutBack<<<gridBlock, threadBlock>>>(d_tetT,d_vertT,d_vertMem,d_vertMemOutside,d_BlockSizes,d_ActiveList,m_maxNumInVert,m_maxNumTotalTets,m_maxVertMappingInside,m_maxVertMappingOutside);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
CopyOutBack<<<gridBlock, threadBlock>>>(d_tetT,d_vertT,d_vertMem,d_vertMemOutside,d_BlockSizes,d_ActiveList,m_maxNumInVert,m_maxNumTotalTets,m_maxVertMappingInside,m_maxVertMappingOutside);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}