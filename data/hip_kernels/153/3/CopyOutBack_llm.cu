#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float3 operator % (const float3 & v1, const float3 & v2)
{   
    return make_float3(v1.y*v2.z - v1.z*v2.y,
                       v1.z*v2.x - v1.x*v2.z,
                       v1.x*v2.y - v1.y*v2.x);
}

__global__ void CopyOutBack(float4* d_tetT, float* d_vertT, int* d_vertMem, int* d_vertMemOutside, int* d_BlockSizes, int* d_ActiveList, int m_maxNumInVert, int m_maxNumTotalTets, int m_maxVertMappingInside, int m_maxVertMappingOutside)
{
    int list_idx = blockIdx.y * gridDim.x + blockIdx.x;
    // retrieve actual block index from the active list
    int block_idx = d_ActiveList[list_idx];
    int block_size = d_BlockSizes[block_idx];

    // Initialize shared memory
    int tx = threadIdx.x;
    int tet_base = block_idx * m_maxNumTotalTets;
    int vert_base = block_idx * m_maxNumInVert;

    if (tx < block_size)
    {
        float T = d_vertT[vert_base + tx];

        // Merge consecutive memory updates and eliminate redundant computations
        int j = 0;
        int tmpindex;
        
        while (j < m_maxVertMappingInside && ((tmpindex = d_vertMem[block_idx * m_maxVertMappingInside * m_maxNumInVert + tx * m_maxVertMappingInside + j]) - tet_base * 4) > -1)
        {
            int segment = tmpindex / 4;
            int offset = tmpindex % 4;
            
            float4& tet = d_tetT[segment];
            switch(offset)
            {
                case 0: tet.x = T; break;
                case 1: tet.y = T; break;
                case 2: tet.z = T; break;
                case 3: tet.w = T; break;
            }
            j++;
        }

        for (int j = 0; j < m_maxVertMappingOutside; j++)
        {
            tmpindex = d_vertMemOutside[block_idx * m_maxVertMappingOutside * m_maxNumInVert + tx * m_maxVertMappingOutside + j];
            if (tmpindex <= -1) break;
            
            int segment = tmpindex / 4;
            int offset = tmpindex % 4;
            
            float4& tet = d_tetT[segment];
            switch(offset)
            {
                case 0: tet.x = T; break;
                case 1: tet.y = T; break;
                case 2: tet.z = T; break;
                case 3: tet.w = T; break;
            }
        }
    }
}