#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void run_reduction(bool *con, bool *blockCon, int* ActiveList, int nActiveBlock, int* blockSizes) {
    int list_idx = blockIdx.y * gridDim.x + blockIdx.x;
    int maxblocksize = blockDim.x;
    int tx = threadIdx.x;
    int block_idx = ActiveList[list_idx];
    int blocksize = blockSizes[block_idx];

    __shared__ bool s_block_conv;

    // Initialize shared memory using the first thread in the block
    if (tx == 0) {
        s_block_conv = true;
    }
    __syncthreads();

    // Perform checks on all participating threads
    if (tx < blocksize) {
        if (!con[maxblocksize * block_idx + tx]) {
            atomicAnd(&s_block_conv, false); // Use atomic operation for shared memory
        }
    }
    __syncthreads();

    // Write the result to global memory
    if (tx == 0) {
        blockCon[block_idx] = s_block_conv;
    }
}