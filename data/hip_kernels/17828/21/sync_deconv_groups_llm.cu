#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void sync_deconv_groups() {
    // Use __syncthreads() to ensure synchronization within the block
    __syncthreads();
}