#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Optimize by assigning unique thread ID for state initialization
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, id, 0, &state[id]);
}