#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Loop unrolling and stride optimization
    for (int stride = idx; stride < num_elements; stride += blockDim.x * gridDim.x) {
        int base_index = stride * 3;
        to_normalize[base_index] = batch_index;
        to_normalize[base_index + 1] = class_index;
        to_normalize[base_index + 2] = static_cast<int64_t>(original[stride]);
    }
}