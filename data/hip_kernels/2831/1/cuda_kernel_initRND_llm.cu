#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_kernel_initRND(unsigned long seed, hiprandState *States)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim.x for better scalability
    hiprand_init(seed, id, 0, &States[id]); // Directly use 'id' for initialization
}