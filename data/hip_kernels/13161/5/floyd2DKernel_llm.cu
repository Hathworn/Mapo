#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void floyd2DKernel(int * M, const int nverts, const int k) {
    // Calculate global thread indices directly, avoiding extra calculations
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Proceed only if within matrix bounds
    if (i < nverts && j < nverts) {
        if (i != j && i != k && j != k) {
            // Optimize memory accesses using shared memory pattern
            int ik = (i * nverts) + k;
            int kj = (k * nverts) + j;
            int ij = (i * nverts) + j;

            // Compute new path cost and update if better
            int aux = M[ik] + M[kj];
            if (aux < M[ij]) {
                M[ij] = aux;
            }
        }
    }
}