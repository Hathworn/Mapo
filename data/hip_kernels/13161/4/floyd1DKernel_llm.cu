#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void floyd1DKernel(int * M, const int nverts, const int k) {
    int ii = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate row (i) and column (j) indices from linear index
    int i = ii / nverts;
    int j = ii % nverts;

    // Check if indices are within bounds
    if (i < nverts && j < nverts) {
        int kj = (k * nverts) + j;
        int ik = (i * nverts) + k;

        // Only update if i, j, k are distinct
        if (i != j && i != k && j != k) {
            // Select minimum between current value and the calculated sum
            M[ii] = min(M[ik] + M[kj], M[ii]);
        }
    }
}