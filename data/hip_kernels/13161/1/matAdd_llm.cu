#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matAdd(float *A, float *B, float *C, int N) {
    // Calculate the global thread index for both rows and columns
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Row index
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Column index

    // Combined index within the flattened matrix
    if (i < N && j < N) {
        int tid = (i * N) + j;
        C[tid] = A[tid] + B[tid];
    }
}