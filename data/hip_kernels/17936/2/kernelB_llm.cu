#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index

    if (idx < size) { // Check if index is within the range
        r[idx] = x[idx] * y[idx] + z[idx]; // Perform computation
    }
}