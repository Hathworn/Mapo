#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelA(float* r, float* x, float* y, float* z, int size)
{
    // Calculate global index for current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within bounds
    if (i < size) {
        r[i] = x[i] * y[i] + z[i];
    }
}
```
