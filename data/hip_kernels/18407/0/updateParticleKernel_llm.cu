#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const float REAL_VALUE_MAX = 1000000.0f;
const int NUM_THREADS = 32;
const int SIZE = 10000;
const int DIMENSION = 2;

__device__ float clamp(float v, float mn = -REAL_VALUE_MAX, float mx = REAL_VALUE_MAX) {
    return v < mn ? mn : v > mx ? mx : v;
}

__global__ void updateParticleKernel(float* P, float* V, float* PB, float* GB, float momentum, float introvert, float extrovert, float clamp_min, float clamp_max) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= SIZE * DIMENSION) return; // Early exit for out-of-bounds indices

    // Pre-compute reused indices to minimize redundant calculations
    int dimIdx = idx % DIMENSION;

    // Optimize memory access by using registers
    float p = P[idx];
    float v = V[idx];
    float pb = PB[idx];
    float gb = GB[dimIdx];

    // Update particle position and velocity with clamping
    p = clamp(p + v, clamp_min, clamp_max);
    v = clamp(momentum * v + introvert * (pb - p) + extrovert * (gb - p), clamp_min, clamp_max);

    // Write back results
    P[idx] = p;
    V[idx] = v;
}