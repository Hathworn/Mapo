#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void sync_conv_groups() {
    // Use __syncthreads to synchronize threads in the block
    __syncthreads();
}