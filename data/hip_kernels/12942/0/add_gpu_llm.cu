#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 10

__global__ void add_gpu(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}