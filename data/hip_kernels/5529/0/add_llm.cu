#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(std::size_t n, const float *x, float *y) {
    std::size_t index = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global index
    std::size_t stride = blockDim.x * gridDim.x;                // Calculate grid stride

    if (index < n) {                                            // Ensure index within bounds
        y[index] = x[index] + y[index];                         // Perform addition
        index += stride;
    }
}