#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate the global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the total number of threads
    const int numThreads = blockDim.x * gridDim.x;
    
    // Loop over each element assigned to this thread
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Calculate coordinates on the target matrix
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        // Calculate the corresponding coordinates on the source matrix
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        
        // Write the value from the source matrix to the target matrix
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}