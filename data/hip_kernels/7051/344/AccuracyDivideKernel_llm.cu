#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Check to ensure the current thread can process the input
    if (idx < N) {
        accuracy[idx] /= N;
    }
}