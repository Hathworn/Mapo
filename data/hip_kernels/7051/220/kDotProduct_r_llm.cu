#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float temp = 0.0f;

    // Accumulate results to temp to reduce shared memory bank conflicts
    if (eidx < numElements) {
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            temp += a[eidx] * b[eidx];
        }
    }
    shmem[threadIdx.x] = temp;
    __syncthreads();

    // Use unrolling for faster reduction in shared memory
    if (threadIdx.x < 256) shmem[threadIdx.x] += shmem[threadIdx.x + 256];
    __syncthreads();
    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    __syncthreads();

    // Utilize warp shuffle for final reduction within a warp
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem; // Access shared memory as volatile to prevent compiler optimization issues
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    // Only the first thread writes the result back
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}