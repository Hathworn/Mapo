#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Optimize by reducing calculations inside hiprand_init
    const uint offset = 0;
    // Use tidx directly without additional index computation
    hiprand_init(seed, tidx, offset, &state[tidx]);
}