#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Using shared memory for synchronization
    __shared__ float shared_accuracy;

    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy;
    }
    __syncthreads();

    // Only one thread divides and writes back to global memory
    if (threadIdx.x == 0) {
        shared_accuracy /= N;
        *accuracy = shared_accuracy;
    }
}