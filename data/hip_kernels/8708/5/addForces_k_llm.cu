#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addForces_k(float2 *v, int dx, int dy, int spx, int spy, float fx, float fy, int r, size_t pitch) {

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // Calculate the target address once and assign directly
    float2 *fj = (float2*)((char*)v + (ty + spy) * pitch) + tx + spx;
    
    float2 vterm = *fj;
    
    // Move subtraction out of multiply to reduce operations
    int offsetX = tx - r;
    int offsetY = ty - r;
    
    // Simplify the computation of s by combining power calculations
    float s = 1.f / (1.f + (offsetX * offsetX) * (offsetX * offsetX) + (offsetY * offsetY) * (offsetY * offsetY));
    
    // Inline operations for better performance
    vterm.x += s * fx;
    vterm.y += s * fy;
    
    *fj = vterm;
}