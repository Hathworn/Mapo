#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int N)
{
    // Compute global index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Use shared memory for faster access
    extern __shared__ float shared_A[];
    extern __shared__ float shared_B[];
    
    // Load data from global to shared memory
    if (i < N) {
        shared_A[threadIdx.x] = A[i]; 
        shared_B[threadIdx.x] = B[i]; 
    }
    __syncthreads();

    // Ensure threads have loaded data before computation
    if (i < N)
        C[i] = shared_A[threadIdx.x] + shared_B[threadIdx.x];
}