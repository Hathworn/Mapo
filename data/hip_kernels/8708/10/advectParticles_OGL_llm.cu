#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void advectParticles_OGL(float2 *part, float2 *v, int dx, int dy, float dt, int lb, size_t pitch) {
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;

    // Early exit if thread is outside domain in X
    if (gtidx >= dx) return;
    
    for (int p = 0; p < lb; p++) {
        // fi is the domain location in y for this thread
        int fi = gtidy + p;
        if (fi < dy) {
            int fj = fi * dx + gtidx;
            float2 pterm = part[fj];

            // Calculate indices for velocity lookup
            int xvi = min(max((int)(pterm.x * dx), 0), dx - 1);
            int yvi = min(max((int)(pterm.y * dy), 0), dy - 1);

            // Access velocity using pitch
            float2 vterm = *((float2*)((char*)v + yvi * pitch) + xvi);

            // Update particle position
            pterm.x = fmodf(pterm.x + dt * vterm.x + 1.f, 1.f);
            pterm.y = fmodf(pterm.y + dt * vterm.y + 1.f, 1.f);

            part[fj] = pterm;
        }
    }
}