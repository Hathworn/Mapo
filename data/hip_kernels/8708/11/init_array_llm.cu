#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize memory access by using local variable
    int local_factor = *factor;

    for (int i = 0; i < num_iterations; i++)
    {
        g_data[idx] += local_factor;
    }
}