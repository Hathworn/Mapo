#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateVelocity_k(float2 *v, float *vx, float *vy, int dx, int pdx, int dy, int lb, size_t pitch) {
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;

    // Precompute scaling factor outside the loop
    float scale = 1.f / (dx * dy);

    if (gtidx < dx) {
        for (int p = 0; p < lb; p++) {
            int fi = gtidy + p;
            if (fi < dy) {
                int fjr = fi * pdx + gtidx;
                float vxterm = vx[fjr];
                float vyterm = vy[fjr];

                // Precompute the pitch offset
                float2 *fj = (float2*)((char*)v + fi * pitch) + gtidx;

                // Store normalized results directly in the struct
                fj->x = vxterm * scale;
                fj->y = vyterm * scale;
            }
        }
    }
}