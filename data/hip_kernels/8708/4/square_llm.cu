#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(int *array, int arrayCount)
{
    // Improved: Load value into register for faster access and reduced memory traffic
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < arrayCount) {
        int value = array[idx]; // Load once
        array[idx] = value * value; // Compute and store
    }
}