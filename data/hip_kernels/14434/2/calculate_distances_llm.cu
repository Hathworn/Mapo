#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_distances(float * sweeper_pos_v, float * mine_pos_v, int num_sweepers, int num_mines, float * distance_v, float * inputs, int * sweeper_score_v, int width, int height, int size)
{
#define sweeperIdx blockIdx.y
#define mineIdx threadIdx.x*2

int distanceIdx = (blockIdx.y * num_mines) + threadIdx.x;
float vec_x;
float vec_y;
float distance;

__shared__ float sweeper_pos[2];

// Efficient global memory read using one if statement 
if (threadIdx.x < 2)
{
    sweeper_pos[threadIdx.x] = sweeper_pos_v[sweeperIdx + threadIdx.x];
    inputs[((sweeperIdx * 4) + threadIdx.x) + 2] = sweeper_pos[threadIdx.x]; //copy the sweeper position out to the inputs for the neural network in parallel
}

__syncthreads();

// Reduce repetitive access using temporary variables
vec_x = mine_pos_v[mineIdx] - sweeper_pos[0];
vec_y = mine_pos_v[mineIdx + 1] - sweeper_pos[1];
distance = sqrtf((vec_x * vec_x) + (vec_y * vec_y));  // Use sqrtf for better performance
distance_v[distanceIdx] = distance;

// Conditionally increment sweeper scores with improved access patterns
if (distance < size)
{
    mine_pos_v[mineIdx] = ((threadIdx.x + 1) * clock()) % width;
    mine_pos_v[mineIdx + 1] = ((threadIdx.x + 1) * clock()) % height;
    atomicAdd(&sweeper_score_v[sweeperIdx], 1);  // Use atomic operation for score update
}

#undef sweeperIdx
#undef mineIdx
}