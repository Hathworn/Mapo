#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void find_closest_mine(float * mine_pos_v, float * distances_v, int * mineIdx_v, int num_sweeprs, int num_mines, float * inputs)
{
    #define sweeperIdx blockIdx.y
    #define first_item blockIdx.y*num_mines
    int my_index = (gridDim.x * blockIdx.x) + threadIdx.x;

    // Use shared memory for faster access to critical data
    __shared__ float shared_distances_v[1024];
    __shared__ int shared_mineIdx_v[1024];

    if (my_index < num_mines) {
        shared_distances_v[threadIdx.x] = distances_v[first_item + my_index];
        shared_mineIdx_v[threadIdx.x] = my_index;
    }
    __syncthreads();

    // Optimize with shared memory usage
    for (int stride = num_mines / 2; stride > 0; stride /= 2) {
        __syncthreads();
        if (my_index < stride && (my_index + stride) < num_mines) {
            if (shared_distances_v[threadIdx.x] < shared_distances_v[threadIdx.x + stride]) {
                shared_distances_v[threadIdx.x] = shared_distances_v[threadIdx.x + stride];
                shared_mineIdx_v[threadIdx.x] = shared_mineIdx_v[threadIdx.x + stride];
            }
        }
    }

    // After reduction, copy back results to global memory
    if (threadIdx.x == 0) {
        distances_v[first_item] = shared_distances_v[0];
        mineIdx_v[sweeperIdx] = shared_mineIdx_v[0];

        inputs[sweeperIdx * 4] = mine_pos_v[mineIdx_v[sweeperIdx] * 2];
        inputs[sweeperIdx * 4 + 1] = mine_pos_v[mineIdx_v[sweeperIdx] * 2 + 1];
    }
    #undef sweeperIdx
    #undef first_item
}