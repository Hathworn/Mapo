#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void increment_kernel(int *g_data, int inc_value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within the bounds to prevent memory access errors
    if (idx < gridDim.x * blockDim.x) {
        g_data[idx] += inc_value; // Simplified increment operation
    }
}