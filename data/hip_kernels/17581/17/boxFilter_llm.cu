#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void boxFilter(unsigned char *srcImage, unsigned char *dstImage, unsigned int width, unsigned int height, int channel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate filter offsets
    const int filter_offset_x = FILTER_WIDTH / 2;
    const int filter_offset_y = FILTER_HEIGHT / 2;

    // Threads inside image boundary will perform filter
    if (x >= filter_offset_x && x < (width - filter_offset_x) && y >= filter_offset_y && y < (height - filter_offset_y))
    {
        for (int c = 0; c < channel; c++)
        {
            float sum = 0.0f;
            // Use constant summation of filter area
            const float kS = FILTER_WIDTH * FILTER_HEIGHT;

            // Sum pixel values inside filter
            for (int ky = -filter_offset_y; ky <= filter_offset_y; ky++)
            {
                for (int kx = -filter_offset_x; kx <= filter_offset_x; kx++)
                {
                    float fl = srcImage[((y + ky) * width + (x + kx)) * channel + c];
                    sum += fl;
                }
            } 
            
            // Store averaged value
            dstImage[(y * width + x) * channel + c] = sum / kS;
        } 
    }
}