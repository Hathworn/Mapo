#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void remove_redness_from_coordinates(const unsigned int* d_coordinates, unsigned char* d_r, unsigned char* d_b, unsigned char* d_g, unsigned char* d_r_output, int num_coordinates, int num_pixels_y, int num_pixels_x, int template_half_height, int template_half_width)
{
    // Calculate global thread index
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    // Get total image size
    int imgSize = num_pixels_x * num_pixels_y;

    // Proceed if within the range of coordinates
    if (global_index_1d < num_coordinates)
    {
        // Calculate 1D and 2D image indices
        unsigned int image_index_1d = d_coordinates[imgSize - global_index_1d - 1];
        ushort2 image_index_2d = make_ushort2(image_index_1d % num_pixels_x, image_index_1d / num_pixels_x);

        // Loop through template region surrounding the pixel
        for (int y = image_index_2d.y - template_half_height; y <= image_index_2d.y + template_half_height; y++)
        {
            for (int x = image_index_2d.x - template_half_width; x <= image_index_2d.x + template_half_width; x++)
            {
                // Clamp coordinates to image boundaries
                int x_clamped = min(num_pixels_x - 1, max(0, x));
                int y_clamped = min(num_pixels_y - 1, max(0, y));

                // Calculate clamped 1D index
                int image_offset_index_1d_clamped = y_clamped * num_pixels_x + x_clamped;

                // Fetch green and blue values
                unsigned char g_value = d_g[image_offset_index_1d_clamped];
                unsigned char b_value = d_b[image_offset_index_1d_clamped];

                // Calculate average and update red channel output
                d_r_output[image_offset_index_1d_clamped] = (unsigned char)((g_value + b_value) / 2);
            }
        }
    }
}