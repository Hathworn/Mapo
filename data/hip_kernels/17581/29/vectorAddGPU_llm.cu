#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAddGPU(float *a, float *b, float *c, int N)
{
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the index is within bounds
    if (idx < N) 
    {
        // Perform vector addition directly
        c[idx] = a[idx] + b[idx];
    }
}