#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_matrix_transpose(int* mat_in, int* mat_out, unsigned int rows, unsigned int cols)
{
    // Calculate global thread row and column indices
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds and perform the transpose
    if (idx < cols && idy < rows)
    {
        unsigned int pos = idy * cols + idx;
        unsigned int trans_pos = idx * rows + idy;
        mat_out[trans_pos] = mat_in[pos];
    }
}