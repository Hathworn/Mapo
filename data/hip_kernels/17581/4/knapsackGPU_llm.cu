#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void knapsackGPU(int* dp, int row, int* d_value, int* d_weight, int capacity)
{
    // Calculate global index
    int in = threadIdx.x + (blockDim.x * blockIdx.x);
    
    // Calculate index for dp array
    int ind = in + (row * (capacity + 1));

    // Initialize dp array when row is 0
    if (row == 0) {
        if (in <= capacity) {
            dp[in] = 0;
        }
        return;
    }

    // Perform check within bounds of capacity
    if (in > 0 && in <= capacity) {
        if (in >= d_weight[row - 1]) {
            // Update dp[ind] based on comparison
            dp[ind] = max(dp[ind - (capacity + 1)], 
                          d_value[row - 1] + dp[ind - (capacity + 1) - d_weight[row - 1]]);
        } else {
            dp[ind] = dp[ind - (capacity + 1)];
        }
    } else if (in == 0) {
        dp[ind] = 0;
    }
}