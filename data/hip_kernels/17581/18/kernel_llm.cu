#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float compute(float val) {
    return sqrt(pow(3.14159, val)); // Pre-compute result in device function to reduce redundancy
}

__global__ void kernel(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Calculating stride outside loop
    for (int i = tid; i < n; i += stride) {
        x[i] = compute(i); // Call the pre-computed device function
    }
}