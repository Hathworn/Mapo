#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define FILTER_SIZE (FILTER_WIDTH * FILTER_HEIGHT)

__device__ void sort(unsigned char* filterVector) {
    // Simple insertion sort for the small filter size
    for (int i = 1; i < FILTER_SIZE; ++i) {
        unsigned char key = filterVector[i];
        int j = i - 1;
        while (j >= 0 && filterVector[j] > key) {
            filterVector[j + 1] = filterVector[j];
            j--;
        }
        filterVector[j + 1] = key;
    }
}

__global__ void medianFilter(unsigned char *srcImage, unsigned char *dstImage, unsigned int width, unsigned int height, int channel) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Only threads inside image will write results
    if ((x >= FILTER_WIDTH / 2) && (x < (width - FILTER_WIDTH / 2)) && 
        (y >= FILTER_HEIGHT / 2) && (y < (height - FILTER_HEIGHT / 2))) {
        for (int c = 0; c < channel; c++) {
            unsigned char filterVector[FILTER_SIZE];

            // Loop inside the filter to average pixel values
            for (int ky = -FILTER_HEIGHT / 2; ky <= FILTER_HEIGHT / 2; ky++) {
                for (int kx = -FILTER_WIDTH / 2; kx <= FILTER_WIDTH / 2; kx++) {
                    int index = (ky + FILTER_HEIGHT / 2) * FILTER_WIDTH + (kx + FILTER_WIDTH / 2);
                    filterVector[index] = srcImage[((y + ky) * width + (x + kx)) * channel + c];
                }
            }

            // Sorting values of filter
            sort(filterVector);
            dstImage[(y * width + x) * channel + c] = filterVector[FILTER_SIZE / 2];
        }
    }
}