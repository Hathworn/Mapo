#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function 
__global__ void knapsackGPU2(int* dp, int* d_value, int* d_weight, int capacity, int n)
{
    int in = threadIdx.x + (blockDim.x * blockIdx.x);

    // Loop unrolling, reducing total iterations for row == 0 and row == 1
    if (in <= (capacity + 1)) {
        dp[in] = 0; // Initialize base row to zero
    }

    for (int row = 1; row <= n; row++) 
    {
        int ind = in + (row * (capacity + 1));

        // Conditional check to omit unnecessary executions
        if (in > 0 && in <= (capacity + 1)) 
        {
            if (in >= d_weight[row - 1]) 
            {
                int value_with_item = d_value[row - 1] + dp[ind - (capacity + 1) - d_weight[row - 1]];
                int value_without_item = dp[ind - (capacity + 1)];
                dp[ind] = max(value_with_item, value_without_item); // Simplify to single max function call
            } 
            else 
            {
                dp[ind] = dp[ind - (capacity + 1)];
            }
        }
    }
}