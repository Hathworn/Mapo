#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void laplacianFilter(unsigned char *srcImage, unsigned char *dstImage, unsigned int width, unsigned int height)
{
    // Calculate 2D index from block and thread indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Define constant kernel to improve performance
    __constant__ float ker[3][3] = {{0, -1, 0}, {-1, 4, -1}, {0, -1, 0}};

    // Check if thread is within the bounds of the image
    if ((x >= 1) && (x < width - 1) && (y >= 1) && (y < height - 1))
    {
        // Use local register to store pixel sum
        float sum = 0;

        // Unroll loop for fixed kernel size for better performance
        for (int ky = -1; ky <= 1; ky++)
        {
            for (int kx = -1; kx <= 1; kx++)
            {
                float fl = srcImage[((y + ky) * width + (x + kx))];
                sum += fl * ker[ky + 1][kx + 1];
            }
        }
        // Store the result in the destination image
        dstImage[(y * width + x)] = sum;
    }
}