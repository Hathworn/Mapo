#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void naive_normalized_cross_correlation(float* d_response, unsigned char* d_original, unsigned char* d_template, int num_pixels_y, int num_pixels_x, int template_half_height, int template_height, int template_half_width, int template_width, int template_size, float template_mean)
{
    int nx = num_pixels_x;
    int knx = template_width;
    int2 image_index_2d = make_int2((blockIdx.x * blockDim.x) + threadIdx.x, (blockIdx.y * blockDim.y) + threadIdx.y);
    int image_index_1d = (nx * image_index_2d.y) + image_index_2d.x;

    if (image_index_2d.x < nx && image_index_2d.y < num_pixels_y)
    {
        // Precompute clamped offsets and use shared memory for speed-up
        __shared__ unsigned char s_original[1024];  // Assuming a blockDim of 32x32
        __shared__ unsigned char s_template[1024];  // Assuming a blockDim of 32x32
        int shared_index = threadIdx.y * blockDim.x + threadIdx.x;

        float image_sum = 0.0f;
        for (int y = -template_half_height; y <= template_half_height; ++y)
        {
            for (int x = -template_half_width; x <= template_half_width; ++x)
            {
                int2 image_offset_index_2d = make_int2(image_index_2d.x + x, image_index_2d.y + y);
                int2 image_offset_index_2d_clamped = make_int2(min(nx - 1, max(0, image_offset_index_2d.x)), min(num_pixels_y - 1, max(0, image_offset_index_2d.y)));
                int image_offset_index_1d_clamped = (nx * image_offset_index_2d_clamped.y) + image_offset_index_2d_clamped.x;

                s_original[shared_index] = d_original[image_offset_index_1d_clamped];
                __syncthreads();
                image_sum += (float)s_original[shared_index];
            }
        }

        float image_mean = image_sum / (float)template_size;

        float sum_of_image_template_diff_products = 0.0f;
        float sum_of_squared_image_diffs = 0.0f;
        float sum_of_squared_template_diffs = 0.0f;

        for (int y = -template_half_height; y <= template_half_height; ++y)
        {
            for (int x = -template_half_width; x <= template_half_width; ++x)
            {
                int2 image_offset_index_2d = make_int2(image_index_2d.x + x, image_index_2d.y + y);
                int2 image_offset_index_2d_clamped = make_int2(min(nx - 1, max(0, image_offset_index_2d.x)), min(num_pixels_y - 1, max(0, image_offset_index_2d.y)));
                int image_offset_index_1d_clamped = (nx * image_offset_index_2d_clamped.y) + image_offset_index_2d_clamped.x;

                // Using shared memory for template as well
                int2 template_index_2d = make_int2(x + template_half_width, y + template_half_height);
                int template_index_1d = (knx * template_index_2d.y) + template_index_2d.x;

                s_template[shared_index] = d_template[template_index_1d];
                __syncthreads();

                float image_diff = (float)s_original[shared_index] - image_mean;
                float template_diff = s_template[shared_index] - template_mean;

                float image_template_diff_product = s_original[shared_index] * template_diff;
                float squared_image_diff = image_diff * image_diff;
                float squared_template_diff = template_diff * template_diff;

                sum_of_image_template_diff_products += image_template_diff_product;
                sum_of_squared_image_diffs += squared_image_diff;
                sum_of_squared_template_diffs += squared_template_diff;
            }
        }

        // Compute final result
        float result_value = 0.0f;
        if (sum_of_squared_image_diffs != 0 && sum_of_squared_template_diffs != 0)
        {
            result_value = sum_of_image_template_diff_products / sqrt(sum_of_squared_image_diffs * sum_of_squared_template_diffs);
        }

        d_response[image_index_1d] = result_value;
    }
}