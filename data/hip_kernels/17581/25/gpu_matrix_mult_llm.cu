#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_matrix_mult(int *a, int *b, int *c, int m, int n, int k)
{
    // Calculate row and column indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within bounds of matrix dimensions
    if (row < m && col < k)
    {
        int sum = 0;
        
        // Perform matrix multiplication using shared memory to improve access speed
        extern __shared__ int shared[];
        int* shared_a = &shared[0];
        int* shared_b = &shared[n * blockDim.x];

        for (int i = 0; i < n; i += blockDim.x)
        {
            // Load data into shared memory
            if (row < m && (i + threadIdx.x) < n)
                shared_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * n + i + threadIdx.x];
            else
                shared_a[threadIdx.y * blockDim.x + threadIdx.x] = 0;

            if (col < k && (i + threadIdx.y) < n)
                shared_b[threadIdx.y * blockDim.x + threadIdx.x] = b[(i + threadIdx.y) * k + col];
            else
                shared_b[threadIdx.y * blockDim.x + threadIdx.x] = 0;
            
            // Synchronize to ensure all shared memory loads are complete
            __syncthreads();
            
            // Summation in shared memory
            for (int j = 0; j < blockDim.x; ++j)
            {
                sum += shared_a[threadIdx.y * blockDim.x + j] * shared_b[j * blockDim.x + threadIdx.x];
            }
            
            // Synchronize to ensure all threads have summed
            __syncthreads();
        }

        // Write result to output matrix
        c[row * k + col] = sum;
    }
}