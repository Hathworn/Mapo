#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_add(int* a, int* b, int* c)
{
    // Optimize index calculation by using a register variable
    register int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Use prefetch for a and b to improve memory access
    register int val_a = __ldg(&a[index]);
    register int val_b = __ldg(&b[index]);

    // Store the result directly
    c[index] = val_a + val_b;
}