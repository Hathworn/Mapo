#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16

__global__ void gpu_square_matrix_mult(int *d_a, int *d_b, int *d_result, int n) {
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;

    for (int sub = 0; sub < gridDim.x; ++sub) {
        // Load tiles into shared memory with bounds check
        if (row < n && (sub * BLOCK_SIZE + threadIdx.x) < n) {
            tile_a[threadIdx.y][threadIdx.x] = d_a[row * n + sub * BLOCK_SIZE + threadIdx.x];
        } else {
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }

        if (col < n && (sub * BLOCK_SIZE + threadIdx.y) < n) {
            tile_b[threadIdx.y][threadIdx.x] = d_b[(sub * BLOCK_SIZE + threadIdx.y) * n + col];
        } else {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }

        __syncthreads();

        // Compute partial results
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        
        __syncthreads();
    }

    // Write result to global memory
    if (row < n && col < n) {
        d_result[row * n + col] = tmp;
    }
}