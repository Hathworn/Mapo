#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sharpeningFilter(unsigned char* srcImage, unsigned char* dstImage, unsigned int width, unsigned int height, int channel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    __shared__ float kernel[FILTER_WIDTH][FILTER_HEIGHT]; // Use shared memory for kernel
    if (threadIdx.x < FILTER_WIDTH && threadIdx.y < FILTER_HEIGHT) {
        kernel[threadIdx.y][threadIdx.x] = (threadIdx.y == 1 && threadIdx.x == 1) ? 9.0f : -1.0f; // Initialize kernel in shared memory
    }
    __syncthreads(); // Ensure kernel is loaded in shared memory

    // Check if thread is within bounds of the image
    if ((x >= FILTER_WIDTH / 2) && (x < (width - FILTER_WIDTH / 2)) && (y >= FILTER_HEIGHT / 2) && (y < (height - FILTER_HEIGHT / 2)))
    {
        for (int c = 0; c < channel; c++)
        {
            float sum = 0.0f;
            // Convolution using the kernel
            for (int ky = -FILTER_HEIGHT / 2; ky <= FILTER_HEIGHT / 2; ky++) {
                for (int kx = -FILTER_WIDTH / 2; kx <= FILTER_WIDTH / 2; kx++) {
                    float fl = srcImage[((y + ky) * width + (x + kx)) * channel + c];
                    sum += fl * kernel[ky + FILTER_HEIGHT / 2][kx + FILTER_WIDTH / 2];
                }
            }
            // Save computed pixel value
            dstImage[(y * width + x) * channel + c] = min(max(sum, 0.0f), 255.0f); // Clamp result to valid range
        }
    }
}