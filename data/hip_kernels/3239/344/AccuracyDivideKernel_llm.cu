#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Check if the index is zero for a single-threaded operation
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}