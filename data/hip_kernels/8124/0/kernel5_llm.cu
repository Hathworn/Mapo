#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

typedef float dtype;

#define N_ (8 * 1024 * 1024)
#define MAX_THREADS 256 // threads per block
#define MAX_BLOCKS 64

#define MIN(x,y) ((x < y) ? x : y)

/* return the next power of 2 number that is larger than x */
__global__ void kernel5(dtype *g_idata, dtype *g_odata, unsigned int n)
{
    __shared__ volatile dtype scratch[MAX_THREADS];
    unsigned int bid = gridDim.x * blockIdx.y + blockIdx.x;
    unsigned int blockDimNew = blockDim.x * 2;
    unsigned int i = (bid * blockDimNew) + threadIdx.x;
    unsigned int gridSize = blockDim.x * 2 * gridDim.x;

    // Initialize scratch array to 0
    scratch[threadIdx.x] = 0;
    
    // Efficient strided access for reading
    while(i < n) {
        scratch[threadIdx.x] += g_idata[i] + (i + blockDim.x < n ? g_idata[i + blockDim.x] : 0);
        i += gridSize;
    }
    __syncthreads();

    // Reduction loop unrolling and remove unnecessary synchronization
    for(int stride = (blockDim.x / 2); stride > 0; stride >>= 1) {
        if(threadIdx.x < stride) {
            scratch[threadIdx.x] += scratch[threadIdx.x + stride];
        }
        __syncthreads(); 
    }

    // Write result from each block to global memory
    if(threadIdx.x == 0) {
        g_odata[bid] = scratch[0];
    }
}