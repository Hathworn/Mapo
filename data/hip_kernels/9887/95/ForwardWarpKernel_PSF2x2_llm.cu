#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Cache values in shared memory to reduce redundant memory access
    float cx = u[flow_row_offset + j] * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + static_cast<float>(i) + 1.0f;
   
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);
    float value = src[image_row_offset + j];
    float weight;

    // Utilize shared memory for storing partial sums to reduce memory contention
    __shared__ float sharedDst[1024]; // Assuming block size fits within 1024 shared memory
    __shared__ float sharedNorm[1024];
    int sharedOffset = threadIdx.y * blockDim.x + threadIdx.x;
    sharedDst[sharedOffset] = 0;
    sharedNorm[sharedOffset] = 0;
    __syncthreads();

    // Process all four possible target pixels
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        weight = dx * dy;
        atomicAdd(&sharedDst[sharedOffset], value * weight);
        atomicAdd(&sharedNorm[sharedOffset], weight);
    }

    tx -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        weight = (1.0f - dx) * dy;
        atomicAdd(&sharedDst[sharedOffset], value * weight);
        atomicAdd(&sharedNorm[sharedOffset], weight);
    }

    ty -= 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(&sharedDst[sharedOffset], value * weight);
        atomicAdd(&sharedNorm[sharedOffset], weight);
    }

    tx += 1;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        weight = dx * (1.0f - dy);
        atomicAdd(&sharedDst[sharedOffset], value * weight);
        atomicAdd(&sharedNorm[sharedOffset], weight);
    }

    // Write back accumulated results from shared memory to global memory
    __syncthreads();
    if (sharedDst[sharedOffset] != 0) {
        atomicAdd(&dst[ty * image_stride + tx], sharedDst[sharedOffset]);
    }
    if (sharedNorm[sharedOffset] != 0) {
        atomicAdd(&normalization_factor[ty * image_stride + tx], sharedNorm[sharedOffset]);
    }
}