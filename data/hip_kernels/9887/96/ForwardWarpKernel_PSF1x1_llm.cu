#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < h && j < w) {  // Efficient boundary check
        int flow_row_offset = i * flow_stride + j;
        int image_row_offset = i * image_stride + j;

        float u_ = u[flow_row_offset];
        float v_ = v[flow_row_offset];

        // Calculate coordinates
        float cx = u_ * time_scale + j + 1.0f;
        float cy = v_ * time_scale + i + 1.0f;
        int tx = __float2int_rn(cx);
        int ty = __float2int_rn(cy);

        // Read source pixel
        float value = src[image_row_offset];

        // Check and update destination
        if (tx >= 0 && tx < w && ty >= 0 && ty < h) {  // Simplified condition
            atomicAdd(&dst[ty * image_stride + tx], value);  // Use atomicAdd directly on the pointer
        }
    }
}