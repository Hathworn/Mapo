#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit if out of bounds
    if (ptidx >= npoints) return;

    short2 loc = loc_[ptidx];

    // Optimize memory access by reducing to one indexed operation
    float scaled_x = loc.x * scale;
    float scaled_y = loc.y * scale;

    x[ptidx] = scaled_x;
    y[ptidx] = scaled_y;
}