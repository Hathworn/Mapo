#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global thread index
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;

    // Bound checks to ensure correctness
    if (row < h && col < w) {
        const int pos = row * s + col;
        float scale = normalization_factor[pos];
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);
        image[pos] *= invScale;
    }
}