#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Constants used by the program
#define BLOCK_DIM 16

__global__ void cuInsertionSort(float *dist, long *ind, int width, int height, int k) {
    // Calculate the thread's xIndex
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (xIndex < width) {
        // Pointers to the distance and index arrays
        float *p_dist = dist + xIndex;
        long *p_ind = ind + xIndex;

        // Initialize the first index and max distance
        float max_dist = p_dist[0];
        p_ind[0] = 1;

        // Part 1: Sort the first k elements
        for (int l = 1; l < k; l++) {
            long curr_row = l * width;
            float curr_dist = p_dist[curr_row];
            
            if (curr_dist < max_dist) {
                int i = l - 1;
                while (i >= 0 && p_dist[i * width] > curr_dist) {
                    i--;
                }
                i++;

                // Shift elements to insert the current element
                for (int j = l; j > i; j--) {
                    p_dist[j * width] = p_dist[(j - 1) * width];
                    p_ind[j * width] = p_ind[(j - 1) * width];
                }
                p_dist[i * width] = curr_dist;
                p_ind[i * width] = l + 1;

                // Update max_dist
                max_dist = p_dist[(k - 1) * width];
            } else {
                p_ind[l * width] = l + 1;
            }
        }

        // Part 2: Continue sorting for rest of elements
        for (int l = k; l < height; l++) {
            float curr_dist = p_dist[l * width];
            
            if (curr_dist < max_dist) {
                int i = k - 1;
                while (i >= 0 && p_dist[i * width] > curr_dist) {
                    i--;
                }
                i++;

                // Shift elements to insert the current element
                for (int j = k - 1; j > i; j--) {
                    p_dist[j * width] = p_dist[(j - 1) * width];
                    p_ind[j * width] = p_ind[(j - 1) * width];
                }
                p_dist[i * width] = curr_dist;
                p_ind[i * width] = l + 1;

                // Update max_dist
                max_dist = p_dist[(k - 1) * width];
            }
        }
    }
}