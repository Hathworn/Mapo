#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using blockDim.x and threadIdx.x
__global__ void square(int *array, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        array[tid] *= array[tid];
    }
}