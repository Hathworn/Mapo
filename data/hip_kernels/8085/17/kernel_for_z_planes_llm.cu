#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define OUTSIDE_1 1

__device__ void Device_FloodFillZPlane(int zPlane, int L, int M, int N, unsigned char* vol)
{
    long idx, idxS, idxN, ts;
    bool anyChange = false;
    int x, y;

    ts = L * M * N;
    idx = zPlane * L * M;
    vol[idx] = OUTSIDE_1;

    anyChange = true;
    while(anyChange) {
        anyChange = false;
        // loop from left to right and top to bottom
        for(x = 0; x < L; x++) {
            for(y = 0; y < M; y++) {
                idxS = idx + y * L + x;
                if(vol[idxS] == OUTSIDE_1) {
                    idxN = idxS + L;
                    if((idxN < ts) && (vol[idxN] == 0)) {
                        vol[idxN] = OUTSIDE_1;
                        anyChange = true;
                    }
                    
                    idxN = idxS - L;
                    if((idxN >= 0) && (vol[idxN] == 0)) {
                        vol[idxN] = OUTSIDE_1;
                        anyChange = true;
                    }
                    
                    idxN = idxS + 1;
                    if((idxN < ts) && (vol[idxN] == 0)) {
                        vol[idxN] = OUTSIDE_1;
                        anyChange = true;
                    }
                    
                    idxN = idxS - 1;
                    if((idxN >= 0) && (vol[idxN] == 0)) {
                        vol[idxN] = OUTSIDE_1;
                        anyChange = true;
                    }
                }
            }
        }

        if(anyChange) {
            anyChange = false;
            // loop from right to left and bottom to top
            for(x = L - 1; x >= 0; x--) {
                for(y = M - 1; y >= 0; y--) {
                    idxS = idx + y * L + x;
                    if(vol[idxS] == OUTSIDE_1) {
                        idxN = idxS + L;
                        if((idxN < ts) && (vol[idxN] == 0)) {
                            vol[idxN] = OUTSIDE_1;
                            anyChange = true;
                        }
                        
                        idxN = idxS - L;
                        if((idxN >= 0) && (vol[idxN] == 0)) {
                            vol[idxN] = OUTSIDE_1;
                            anyChange = true;
                        }
                        
                        idxN = idxS + 1;
                        if((idxN < ts) && (vol[idxN] == 0)) {
                            vol[idxN] = OUTSIDE_1;
                            anyChange = true;
                        }
                        
                        idxN = idxS - 1;
                        if((idxN >= 0) && (vol[idxN] == 0)) {
                            vol[idxN] = OUTSIDE_1;
                            anyChange = true;
                        }
                    }
                }
            }
        }
    }
}

__global__ void kernel_for_z_planes(unsigned char *d_vol, int L, int M, int N)
{
    Device_FloodFillZPlane(threadIdx.x, L, M, N, d_vol);
}