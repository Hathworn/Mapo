#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define CHECK_AND_SET(idxN) \
    if((idxN >= 0) && (idxN < ts) && \
    ((vol[idxN] == 0) || (vol[idxN] == OUTSIDE_1) || (vol[idxN] == OUTSIDE_2))) { \
        vol[idxN] = OUTSIDE_3; \
        anyChange = true; \
    }

__device__ void Device_FloodFillXPlane(int xPlane, int L, int M, int N, unsigned char* vol)
{
    long idx, idxS, idxN, ts;
    bool anyChange;
    int y, z;

    ts = L * M * N;
    idx = xPlane;

    // set point (0,0) to OUTSIDE_3
    vol[idx] = OUTSIDE_3;

    anyChange = true;
    while(anyChange) {
        anyChange = false;
        // loop from left to right and top to bottom
        for(y = 0; y < M; y++) {
            for(z = 0; z < N; z++) {
                idxS = z * L * M + L * y + idx;
                if(vol[idxS] == OUTSIDE_3) {
                    // check and set each neighbor
                    CHECK_AND_SET(idxS + L * M)
                    CHECK_AND_SET(idxS - L * M)
                    CHECK_AND_SET(idxS + L)
                    CHECK_AND_SET(idxS - L)
                }
            }
        }

        if(anyChange) {
            anyChange = false;
            // loop from bottom to top and right to left
            for(y = M - 1; y >= 0; y--) {
                for(z = N - 1; z >= 0; z--) {
                    idxS = z * L * M + L * y + idx;
                    if(vol[idxS] == OUTSIDE_3) {
                        CHECK_AND_SET(idxS + L * M)
                        CHECK_AND_SET(idxS - L * M)
                        CHECK_AND_SET(idxS + L)
                        CHECK_AND_SET(idxS - L)
                    }
                }
            }
        }
    }
}

__global__ void kernel_for_x_planes(unsigned char *d_vol, int L, int M, int N) 
{
    Device_FloodFillXPlane(threadIdx.x, L, M, N, d_vol);
}