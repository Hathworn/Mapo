#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void Device_FloodFillYPlane(int yPlane, int L, int M, int N, unsigned char* vol)
{
    long idx, idxS, idxN, ts;
    bool anyChange = false;
    int x, z;

    ts = L * M * N;
    idx = yPlane * L;
    vol[idx] = OUTSIDE_2;

    anyChange = true;
    while (anyChange) {
        anyChange = false;

        // Optimizing loop from left to right and top to bottom
        // Merging both loops into a single pass
        for (x = 0; x < L; x++) {
            for (z = 0; z < N; z++) {
                idxS = z * L * M + idx + x;

                if (vol[idxS] == OUTSIDE_2) {
                    // Check neighbors
                    idxN = idxS + L * M;
                    if (idxN < ts && (vol[idxN] == 0 || vol[idxN] == OUTSIDE_1)) {
                        vol[idxN] = OUTSIDE_2;
                        anyChange = true;
                    }

                    idxN = idxS - L * M;
                    if (idxN >= 0 && (vol[idxN] == 0 || vol[idxN] == OUTSIDE_1)) {
                        vol[idxN] = OUTSIDE_2;
                        anyChange = true;
                    }

                    idxN = idxS + 1;
                    if ((idxS + 1) % L != 0 && (vol[idxN] == 0 || vol[idxN] == OUTSIDE_1)) {
                        vol[idxN] = OUTSIDE_2;
                        anyChange = true;
                    }

                    idxN = idxS - 1;
                    if (idxS % L != 0 && (vol[idxN] == 0 || vol[idxN] == OUTSIDE_1)) {
                        vol[idxN] = OUTSIDE_2;
                        anyChange = true;
                    }
                }
            }
        }

        if (anyChange) {
            anyChange = false;

            // Optimizing loop from bottom to top and right to left
            // Merging both loops into a single pass
            for (x = L - 1; x >= 0; x--) {
                for (z = N - 1; z >= 0; z--) {
                    idxS = z * L * M + idx + x;

                    if (vol[idxS] == OUTSIDE_2) {
                        // Check neighbors
                        idxN = idxS + L * M;
                        if (idxN < ts && (vol[idxN] == 0 || vol[idxN] == OUTSIDE_1)) {
                            vol[idxN] = OUTSIDE_2;
                            anyChange = true;
                        }

                        idxN = idxS - L * M;
                        if (idxN >= 0 && (vol[idxN] == 0 || vol[idxN] == OUTSIDE_1)) {
                            vol[idxN] = OUTSIDE_2;
                            anyChange = true;
                        }

                        idxN = idxS + 1;
                        if ((idxS + 1) % L != 0 && (vol[idxN] == 0 || vol[idxN] == OUTSIDE_1)) {
                            vol[idxN] = OUTSIDE_2;
                            anyChange = true;
                        }

                        idxN = idxS - 1;
                        if (idxS % L != 0 && (vol[idxN] == 0 || vol[idxN] == OUTSIDE_1)) {
                            vol[idxN] = OUTSIDE_2;
                            anyChange = true;
                        }
                    }
                }
            }
        }
    }
}

__global__ void kernel_for_y_planes(unsigned char *d_vol, int L, int M, int N)
{
    Device_FloodFillYPlane(threadIdx.x, L, M, N, d_vol);
}