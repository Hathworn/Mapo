#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/**
*  Project TACO: Parallel ACO algorithm for TSP
*  15-418 Parallel Algorithms - Final Project
*  Ivan Wang, Carl Lin
*/

#define MAX_THREADS 128

__device__ static inline int calculateTo(int i) {
    // Find least triangle number less than i
    int row = (int)(-1 + (sqrt((float)(1 + 8 * i)))) >> 1;
    int tnum = (row * (row + 1)) >> 1;
    int remain = i - tnum;
    return row - remain;
}

__device__ static inline int calculateFrom(int i) {
    // Find least triangle number less than i
    int row = (int)(-1 + (sqrt((float)(1 + 8 * i)))) >> 1;
    int tnum = (row * (row + 1)) >> 1;
    int remain = i - tnum;
    return MAX_CITIES - 1 - remain;
}

__device__ static inline int toIndex(int i, int j) {
    return i * MAX_CITIES + j;
}

__global__ void updateTrails(float *phero, int *paths, float *tourLengths) {
    extern __shared__ float localPaths[]; // Use shared memory to cache paths

    int numPhero = (NUM_EDGES + (blockDim.x * (MAX_ANTS * 2) - 1)) /
                   (blockDim.x * (MAX_ANTS * 2));
    int blockStartPhero = numPhero * blockDim.x * blockIdx.x;
    int from, to;
    
    if (threadIdx.x == 0) {
        // Load paths into shared memory for the current block
        for (int i = 0; i < MAX_ANTS; i++) {
            memcpy(&localPaths[i * MAX_CITIES], &paths[i * MAX_CITIES], MAX_CITIES * sizeof(float));
        }
    }
    __syncthreads();

    int cur_phero;
    for (int i = 0; i < MAX_ANTS; i++) {
        for (int j = 0; j < numPhero; j++) {
            cur_phero = blockStartPhero + j + numPhero * threadIdx.x;

            if (cur_phero >= NUM_EDGES) {
                break;
            }

            from = calculateFrom(cur_phero);
            to = calculateTo(cur_phero);

            bool touched = false;
            int checkTo;
            int checkFrom;
            for (int k = 0; k < MAX_CITIES; k++) {
                checkFrom = localPaths[toIndex(i, k)];
                if (k < MAX_CITIES - 1) {
                    checkTo = localPaths[toIndex(i, k + 1)];
                } else {
                    checkTo = localPaths[toIndex(i, 0)];
                }

                if ((checkFrom == from && checkTo == to) ||
                    (checkFrom == to && checkTo == from)) {
                    touched = true;
                    break;
                }
            }

            if (touched) {
                int idx = toIndex(from, to);
                atomicAdd(&phero[idx], (QVAL / tourLengths[i])); // Use atomic operation for thread safety
                phero[toIndex(to, from)] = phero[idx];
            }
        }
    }
}