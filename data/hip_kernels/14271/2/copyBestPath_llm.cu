#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/**
*  Project TACO: Parallel ACO algorithm for TSP
*  15-418 Parallel Algorithms - Final Project
*  Ivan Wang, Carl Lin
*/

#define MAX_THREADS 128

__global__ void copyBestPath(int i, int *bestPathResult, int *pathResults) {
    // Use optimized memory access by utilizing offset computation
    int offset = i * MAX_ANTS;
    for (int j = threadIdx.x; j < MAX_CITIES; j += blockDim.x) {
        bestPathResult[j] = pathResults[offset + j];
    }
}