#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_THREADS 128

__device__ static inline int toIndex(int i, int j) {
    return i * MAX_CITIES + j;
}

__global__ void updateTrailsAtomic(float *phero, int *paths, float *tourLengths) {
    int antId = blockIdx.x * blockDim.x + threadIdx.x;
    int from, to;

    if (antId < MAX_ANTS) { // Ensure the antId is within bounds
        for (int i = 0; i < MAX_CITIES; i++) {
            from = paths[toIndex(antId, i)];
            if (i < MAX_CITIES - 1) {
                to = paths[toIndex(antId, i+1)];
            } else {
                to = paths[toIndex(antId, 0)];
            }

            if (from < to) {
                int tmp = from;
                from = to;
                to = tmp;
            }
            atomicAdd(&phero[toIndex(from, to)], QVAL / tourLengths[antId]);
        }
    }
}