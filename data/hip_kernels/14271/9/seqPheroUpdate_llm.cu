#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/**
*  Project TACO: Parallel ACO algorithm for TSP
*  15-418 Parallel Algorithms - Final Project
*  Ivan Wang, Carl Lin
*/

#define MAX_THREADS 128

__device__ static inline int toIndex(int i, int j) {
    return i * MAX_CITIES + j;
}

__global__ void seqPheroUpdate(float *phero, float *pheroReal, int *paths, float *tourLengths) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;

    // Use parallel threads to copy pheroReal to phero
    for (int idx = tid; idx < MAX_CITIES * MAX_CITIES; idx += numThreads) {
        phero[idx] = pheroReal[idx];
    }
    __syncthreads();

    if (tid < MAX_CITIES * (MAX_CITIES - 1) / 2) {
        int from = (int)((2 * MAX_CITIES - 1 - sqrtf((2 * MAX_CITIES - 1) * (2 * MAX_CITIES - 1) - 8 * tid)) / 2);
        int to = tid - from * (2 * MAX_CITIES - 1 - from) / 2;
        
        // Evaporate pheromones
        atomicExch(&phero[toIndex(from, to)], phero[toIndex(from, to)] * (1.0 - RHO));
        if (phero[toIndex(from, to)] < 0.0) {
            phero[toIndex(from, to)] = INIT_PHER;
        }
        atomicExch(&phero[toIndex(to, from)], phero[toIndex(from, to)]);
    }
    
    // Add new pheromone to the trails
    for (int ant = tid; ant < MAX_ANTS; ant += numThreads) {
        for (int i = 0; i < MAX_CITIES; i++) {
            int from = paths[toIndex(ant, i)];
            int to = (i < MAX_CITIES - 1) ? paths[toIndex(ant, i+1)] : paths[toIndex(ant, 0)];
            
            float delta = QVAL / tourLengths[ant];
            atomicAdd(&phero[toIndex(from, to)], delta);
            atomicExch(&phero[toIndex(to, from)], phero[toIndex(from, to)]);
        }
    }
}