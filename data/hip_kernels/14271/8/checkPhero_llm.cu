#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_THREADS 128

__device__ static inline int toIndex(int i, int j) {
    return i * MAX_CITIES + j;
}

__global__ void checkPhero(float *pheroSeq, float *phero) {
    // Calculate unique indices for each thread to parallelize comparisons
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = idx / MAX_CITIES;
    int j = idx % MAX_CITIES;
    
    // Ensure person-label uniqueness and prevent self-comparison
    if (i < MAX_CITIES && j < MAX_CITIES && i != j) {
        int globalIdx = toIndex(i, j);
        if (fabsf(pheroSeq[globalIdx] - phero[globalIdx]) > 0.001) {
            printf("PHERO IS BROKEN at (%d, %d); expected: %1.15f, actual: %1.15f\n", i, j, pheroSeq[globalIdx], phero[globalIdx]);
        }
    }
}