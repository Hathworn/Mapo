#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int a, int b, int *c)
{
    // Efficiently compute the sum in parallel
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx == 0) // Ensure only one thread writes the result
    {
        *c = a + b;
    }
}