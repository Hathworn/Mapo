#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c, int N)
{
    int tID = blockIdx.x * blockDim.x + threadIdx.x;  // Use both block and thread index
    if (tID < N) 
    {
        c[tID] = a[tID] + b[tID];  // Efficiently utilize thread parallelism
    }
}