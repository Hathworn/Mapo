#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Use threadIdx and blockIdx for better indexing
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int i = COL * y + x;
    
    // Check boundary condition to avoid out-of-bounds access
    if (x < COL && y < ROW) {
        c[i] = a[i] + b[i];
    }
}
```
