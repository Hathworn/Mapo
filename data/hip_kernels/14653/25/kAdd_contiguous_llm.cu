#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEls) {  // Early exit for threads with no work
        dest[idx] = a[idx] + b[idx];
    }
}