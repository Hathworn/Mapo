#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;  // Calculate stride for multiple iterations
    for (int i = tid; i < N; i += stride) { // Use stride loop to allow for more threads
        out[i] = in[i] > 0 ? in[i] : 0; // Use conditional operator for ReLU
    }
}