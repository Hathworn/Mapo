#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure tid is within bounds and perform conditional assignment
    if (tid < N) {
        ingrad[tid] = (indata[tid] > 0) ? outgrad[tid] : 0.0f;  // Use float literal for clarity
    }
}