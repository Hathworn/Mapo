#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction(int* input, int* output) {
    __shared__ int tmp[TPB];

    // Load elements into shared memory
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    tmp[threadIdx.x] = input[index];
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            tmp[threadIdx.x] += tmp[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Write result for this block to output
    if (threadIdx.x == 0) {
        output[blockIdx.x] = tmp[0];
    }
}