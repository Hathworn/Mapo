#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bigstencil(int* __restrict__ in, int* __restrict__ out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory to reduce global memory accesses
    __shared__ int temp[TILE_SIZE]; // TILE_SIZE should be defined appropriately
    temp[threadIdx.x] = in[i];
    __syncthreads();
    out[i] = temp[threadIdx.x] + 2;
}