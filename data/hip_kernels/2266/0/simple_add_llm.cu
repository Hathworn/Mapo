#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_add(float* __restrict__ a, const float* __restrict__ b, int n)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we don't exceed array bounds
    if (i < n) {
        // Perform vector addition
        a[i] += b[i];
    }
}