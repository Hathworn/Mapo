#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    // Use shared memory for vector_in to reduce global memory access
    __shared__ double shared_vector_in[1024]; // Assumed max blockDim.x of 1024, adjust if necessary

    long i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < dim_mn) {
        double out = 0.0;
        
        // Load vector_in elements into shared memory
        if (threadIdx.x < dim_mn) {
            shared_vector_in[threadIdx.x] = vector_in[threadIdx.x];
        }
        __syncthreads(); // Ensure all data is loaded into shared memory
        
        for (long j = 0; j < dim_mn; j++) {
            out += matrix[i * dim_mn + j] * shared_vector_in[j];
        }
        
        vector_out[i] = out;
    }
}