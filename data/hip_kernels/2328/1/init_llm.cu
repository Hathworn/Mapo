#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

// image size
int rows = 1224, cols = 1624;
int imgSize = rows*cols;

// iterations for stereo matching algorithm
int iteration = 1;

// disparity range
int Dmin = 1;
int Dmax = 80;
int Drange = Dmax - Dmin + 1;
//int winRadius = 9;

// device image pointer
float* dLImgPtr_f = NULL;
float* dRImgPtr_f = NULL;
size_t lPitch, rPitch;

// texture memory for stereo image pair <Type, Dim, ReadMode>
texture<float, 2, hipReadModeElementType> lTex;
texture<float, 2, hipReadModeElementType> rTex;

// timing arrays
const int nt = 2;
double start[nt], end[nt];
double random_start[nt], random_end[nt];
double main_start[nt], main_end[nt];

// evaluate window-based disimilarity
__global__ void init(unsigned int seed, hiprandState_t* states, int cols)
{
    // Calculate unique thread index
    int idx = blockIdx.y * blockDim.y * cols + blockIdx.x * blockDim.x + threadIdx.y * cols + threadIdx.x;
    
    // Initialize RNG state
    hiprand_init(seed, idx, 0, &states[idx]);
}