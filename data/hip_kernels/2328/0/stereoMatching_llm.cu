#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

// image size
int rows = 1224, cols = 1624;
int imgSize = rows * cols;

// iterations for stereo matching algorithm
int iteration = 1;

// disparity range
int Dmin = 1;
int Dmax = 80;
int Drange = Dmax - Dmin + 1;

// device image pointer
float* dLImgPtr_f = NULL;
float* dRImgPtr_f = NULL;
size_t lPitch, rPitch;

// texture memory for stereo image pair <Type, Dim, ReadMode>
texture<float, 2, hipReadModeElementType> lTex;
texture<float, 2, hipReadModeElementType> rTex;

// timing arrays
const int nt = 2;
double start[nt], end[nt];
double random_start[nt], random_end[nt];
double main_start[nt], main_end[nt];

// evaluate window-based disimilarity
__device__ float evaluateCost(float u, float v, float matchIdx, int cols, int rows, int winRadius)
{
    float cost = 0.0f;

    for (int h = -winRadius; h <= winRadius; h++) {
        for (int w = -winRadius; w <= winRadius; w++) {
            cost += fabsf(tex2D(lTex, matchIdx + w / (float)cols, v + h / (float)rows)
                        - tex2D(rTex, u + w / (float)cols, v + h / (float)rows));
        }
    }

    return cost;
}

__global__ void stereoMatching(float* dRDispPtr, float* dRPlanes, int cols, int rows, hiprandState* states, int iteration)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int winRadius = 9;

    // Skip border pixels
    if (x >= cols - winRadius || x < winRadius || y >= rows - winRadius || y < winRadius)
        return;

    float u = x / (float)cols;
    float v = y / (float)rows;
    int idx = y * cols + x;

    // Set planes to be fronto-parallel during first iteration
    if (iteration != 0) {
        dRPlanes[idx * 3] = 0.0f;
        dRPlanes[idx * 3 + 1] = 0.0f;
        dRPlanes[idx * 3 + 2] = 1.0f;
    }

    // Current pixel disparity
    float min_cost = evaluateCost(u, v, u + dRDispPtr[idx] * 80.0f / (float)cols, cols, rows, winRadius);

    // Check left neighbor disparity
    float left_disp = dRDispPtr[idx - 1];
    float left_cost = evaluateCost(u, v, u + left_disp * 80.0f / (float)cols, cols, rows, winRadius);
    if (left_cost < min_cost) {
        min_cost = left_cost;
        dRDispPtr[idx] = left_disp;
    }

    // Check upper neighbor disparity
    float upper_disp = dRDispPtr[idx - cols];
    float upper_cost = evaluateCost(u, v, u + upper_disp * 80.0f / (float)cols, cols, rows, winRadius);
    if (upper_cost < min_cost) {
        min_cost = upper_cost;
        dRDispPtr[idx] = upper_disp;
    }

    // Random disparity within border
    float rand_disp, rand_cost;
    do {
        rand_disp = hiprand_uniform(&states[idx]) * 80.0f;
    } while ((u + rand_disp / (float)cols < (float)winRadius / cols) || (u + rand_disp / (float)cols >= (float)(cols - winRadius) / cols));

    rand_cost = evaluateCost(u, v, u + rand_disp * 80.0f / (float)cols, cols, rows, winRadius);
    if (rand_cost < min_cost) {
        min_cost = rand_cost;
        dRDispPtr[idx] = rand_disp;
    }
}