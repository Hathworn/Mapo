#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "stereoMatching.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *dRDispPtr = NULL;
hipMalloc(&dRDispPtr, XSIZE*YSIZE);
float *dRPlanes = NULL;
hipMalloc(&dRPlanes, XSIZE*YSIZE);
int cols = YSIZE;
int rows = XSIZE;
hiprandState *states = NULL;
hipMalloc(&states, XSIZE*YSIZE);
int iteration = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
stereoMatching<<<gridBlock, threadBlock>>>(dRDispPtr,dRPlanes,cols,rows,states,iteration);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
stereoMatching<<<gridBlock, threadBlock>>>(dRDispPtr,dRPlanes,cols,rows,states,iteration);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
stereoMatching<<<gridBlock, threadBlock>>>(dRDispPtr,dRPlanes,cols,rows,states,iteration);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}