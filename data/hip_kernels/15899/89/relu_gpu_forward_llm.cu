#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Ensure the total threads do not exceed N
    for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < N; tid += blockDim.x * gridDim.x) {
        // Use max function for clarity
        out[tid] = max(in[tid], 0.0f);
    }
}