#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float local_sum = 0.0f;

    // loop to process all required elements
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        local_sum += a[eidx] * b[eidx];
    }
    shmem[threadIdx.x] = local_sum; // store local sum to shared memory
    __syncthreads();

    // unrolled reduction in shared memory
    if (threadIdx.x < 256) shmem[threadIdx.x] += shmem[threadIdx.x + 256];
    __syncthreads();
    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    __syncthreads();
    
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem; // use volatile to prevent compiler optimization issues
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];

        if (threadIdx.x == 0) {
            target[blockIdx.x] = mysh[0];
        }
    }
}