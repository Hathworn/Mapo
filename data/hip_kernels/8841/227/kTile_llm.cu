#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate 1D global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total number of threads
    const int numThreads = blockDim.x * gridDim.x;
    // Loop over elements to process
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Calculate 2D coordinates in the target array
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        // Map to 2D coordinates in the source array using modulo
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Copy data from source to target with calculated indices
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}