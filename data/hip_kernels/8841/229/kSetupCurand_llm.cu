#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Initialize hiprand state for each thread with unique sequence number
    hiprand_init(seed, tidx, 0, &state[tidx]);
}