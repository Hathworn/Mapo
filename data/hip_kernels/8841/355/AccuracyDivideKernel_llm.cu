#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to optimize division
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Check boundary condition in case of multiple blocks/threads
    if(threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}