#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    // Ensure thread index is within the number of cases.
    if (tx < numCases) {
        const int label = static_cast<int>(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Calculate correctProb only when labelp matches maxp
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            // Optimize loop to fetch probs data only once.
            for (int i = 0; i < numOut; i++) {
                float currentProb = probs[i * numCases + tx];
                numMax += (currentProb == maxp);
            }
            // Change division order to avoid unnecessary float conversion.
            correctProbs[tx] = 1.0f / numMax;
        }
    }
}