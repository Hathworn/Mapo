#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _negateStencilKernel(int* stencil, int size, int* out)
{
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Early exit condition moved to reduce divergent branches
    if (idx < size) {
        // Use direct calculation to handle the same logic without a ternary operator
        out[idx] = 1 - stencil[idx];
    }
}