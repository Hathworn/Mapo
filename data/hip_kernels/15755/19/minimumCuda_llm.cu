#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void minimumCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height) 
{
    // Calculate the global x and y position using block and thread indices
    const uint32_t index = blockIdx.y * blockDim.y * width + blockIdx.x * blockDim.x + threadIdx.y * width + threadIdx.x;

    // Check bounds for single flat index
    if (index < width * height) {
        // Pointers for input and output data based on calculated index
        const uint8_t *in1X = in1 + index;
        const uint8_t *in2X = in2 + index;
        uint8_t *outX = out + index;

        // Compare and assign minimum value using ternary operator
        *outX = (*in1X < *in2X) ? *in1X : *in2X;
    }
}