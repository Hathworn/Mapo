#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogramCuda( const uint8_t * data, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * histogram )
{
    // Use shared memory to minimize global memory access
    __shared__ uint32_t localHist[256];
    if (threadIdx.x < 256) {
        localHist[threadIdx.x] = 0;
    }
    __syncthreads();

    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if ( x < width && y < height ) {
        const uint32_t id = y * rowSize + x;
        atomicAdd( &localHist[data[id]], 1 );
    }
    __syncthreads();

    if (threadIdx.x < 256) {
        atomicAdd(&(histogram[threadIdx.x]), localHist[threadIdx.x]);
    }
}