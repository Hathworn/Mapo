#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitwiseXorCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Check boundary conditions
    if ( x < width && y < height ) {
        const uint32_t idx = y * rowSizeOut + x;  // Use one index calculation for efficiency
        out[idx] = in1[y * rowSizeIn1 + x] ^ in2[y * rowSizeIn2 + x];  // Perform XOR operation
    }
}