#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void projectionProfileVerticalCuda(const uint8_t * image, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * projection)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Use shared memory to reduce global memory access
    __shared__ uint32_t localProjection[1024];

    if (threadIdx.y == 0 && x < width)
        localProjection[threadIdx.x] = 0;

    __syncthreads();

    if (x < width && y < height) {
        const uint8_t *imageY = image + y * rowSize + x;
        atomicAdd(&localProjection[threadIdx.x], (*imageY));
    }

    __syncthreads();

    // Write back to global memory
    if (threadIdx.y == 0 && x < width) {
        atomicAdd(&projection[x], localProjection[threadIdx.x]);
    }
}