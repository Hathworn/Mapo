#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void invertCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate global thread ID
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds before inverting the pixel value
    if (x < width && y < height) {
        out[y * rowSizeOut + x] = ~in[y * rowSizeIn + x];
    }
}