#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void absoluteDifferenceCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate the global index once for thread
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Proceed only if within bounds
    if (x < width && y < height) {
        // Use direct access to global memory using flattened index
        const uint32_t indexIn1 = y * rowSizeIn1 + x;
        const uint32_t indexIn2 = y * rowSizeIn2 + x;
        const uint32_t indexOut = y * rowSizeOut + x;

        // Perform the absolute difference calculation
        out[indexOut] = abs(in1[indexIn1] - in2[indexIn2]);
    }
}