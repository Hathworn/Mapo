#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setPixelCuda(uint8_t *in, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t *pointX, uint32_t *pointY, uint32_t pointSize, uint32_t value)
{
    // Calculate the global thread index
    const uint32_t idPoint = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the index is within the point size
    if (idPoint < pointSize)
    {
        // Use register variables to reduce global memory access
        const uint32_t x = pointX[idPoint];
        const uint32_t y = pointY[idPoint];

        // Check if the coordinates are within bounds and set the pixel value
        if (x < width && y < height)
        {
            in[y * rowSize + x] = value;
        }
    }
}