#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtractCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height) 
{
    // Calculate 1D global thread index for better memory coalescing
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < width && idy < height) {
        uint32_t offsetIn1 = idy * rowSizeIn1 + idx;
        uint32_t offsetIn2 = idy * rowSizeIn2 + idx;
        uint32_t offsetOut = idy * rowSizeOut + idx;

        // Use local variables to reduce pointer indirections
        uint8_t valIn1 = in1[offsetIn1];
        uint8_t valIn2 = in2[offsetIn2];
        out[offsetOut] = (valIn1 > valIn2) ? (valIn1 - valIn2) : 0;
    }
}