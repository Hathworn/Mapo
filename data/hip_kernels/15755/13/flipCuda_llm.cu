#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flipCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height, bool horizontal, bool vertical)
{
    const uint32_t inX = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t inY = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure the threads are within image boundaries
    if (inX >= width || inY >= height) return;

    // Use conditional expression to eliminate branching
    const uint32_t outX = horizontal * (width - 1 - inX) + (!horizontal) * inX;
    const uint32_t outY = vertical * (height - 1 - inY) + (!vertical) * inY;

    // Write the flipped pixel to the output image
    out[outY * rowSizeOut + outX] = in[inY * rowSizeIn + inX];
}