#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rotateCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, float inXStart, float inYStart, uint32_t width, uint32_t height, float cosAngle, float sinAngle)
{
    uint32_t outX = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t outY = blockDim.y * blockIdx.y + threadIdx.y;

    // Only if this thread corresponds to a valid pixel in the output
    if (outX < width && outY < height) {
        // Calculate exact input coordinates using rotation
        float exactInX = inXStart + cosAngle * outX + sinAngle * outY;
        float exactInY = inYStart - sinAngle * outX + cosAngle * outY;

        int32_t inX = static_cast<int32_t>(exactInX);
        int32_t inY = static_cast<int32_t>(exactInY);

        // Compute output pixel position
        uint8_t *pixelOut = out + outY * rowSizeOut + outX;

        // Check input boundaries and apply bilinear interpolation if within bounds
        if (inX >= 0 && inX < width - 1 && inY >= 0 && inY < height - 1) {
            const uint8_t *pixelIn = in + inY * rowSizeIn + inX;

            // Calculate interpolation weights
            float probX = exactInX - inX;
            float probY = exactInY - inY;

            // Bilinear interpolation
            float mean = pixelIn[0] * (1 - probX) * (1 - probY) +
                         pixelIn[1] * probX * (1 - probY) +
                         pixelIn[rowSizeIn] * (1 - probX) * probY +
                         pixelIn[rowSizeIn + 1] * probX * probY + 0.5f;

            *pixelOut = static_cast<uint8_t>(mean);
        } else {
            *pixelOut = 0; // Set to 0 if out-of-bounds
        }
    }
}