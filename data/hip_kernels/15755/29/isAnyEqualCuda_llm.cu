#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void isAnyEqualCuda(const uint8_t *image, const uint8_t *value, size_t valueCount, uint32_t width, uint32_t height, uint32_t *differenceCount)
{
    uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Early exit if out of bounds
    if (x >= width || y >= height) return;

    uint32_t id = y * width + x;
    uint8_t pixelValue = image[id];

    // Use shared memory for value array for faster access
    __shared__ uint8_t sharedValues[256]; // Assuming valueCount <= 256, adjust as needed
    if (threadIdx.x < valueCount) {
        sharedValues[threadIdx.x] = value[threadIdx.x];
    }
    __syncthreads();

    // Efficient search using shared memory
    bool equal = false;
    for (uint32_t i = 0; i < valueCount; ++i) {
        if (pixelValue == sharedValues[i]) {
            equal = true;
            break;
        }
    }

    // Atomic add only if equal is true
    if (equal) {
        atomicAdd(differenceCount, 1);
    }
}