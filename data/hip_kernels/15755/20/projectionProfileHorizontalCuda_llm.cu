#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void projectionProfileHorizontalCuda(const uint8_t *image, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t *projection)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockIdx.y;  // Use a single thread per row for better warp utilization

    if (x < width && y < height) {
        const uint8_t *imageRow = image + y * rowSize;  // Pre-compute row start to reduce index calculation
        atomicAdd(&projection[x], imageRow[x]);
    }
}