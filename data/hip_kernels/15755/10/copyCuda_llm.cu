#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
    // Calculate global thread ID for improved memory access
    const uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure within image bounds
    if ( idx < width && idy < height ) {
        const uint32_t inIndex = idy * rowSizeIn + idx;
        const uint32_t outIndex = idy * rowSizeOut + idx;
        // Direct copy from in to out
        out[outIndex] = in[inIndex];
    }
}