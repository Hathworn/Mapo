#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lookupTableCuda(const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height, uint8_t * table)
{
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    // Use shared memory to cache table values if possible (not used here as it's a global lookup)
    if (x < width && y < height) {
        // Lookup using table directly
        out[y * rowSizeOut + x] = table[in[y * rowSizeIn + x]];
    }
}