#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitwiseAndCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
    // Calculate a single thread index
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Boundary check to ensure the thread falls within matrix dimensions
    if ( x < width && y < height ) {
        // Calculate linear memory indexes for input and output
        const uint32_t linearIndex = y * width + x;
        out[linearIndex] = in1[linearIndex] & in2[linearIndex];
    }
}