#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void isEqualCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint32_t width, uint32_t height, uint32_t *isEqual) {
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Use shared memory to reduce global memory access
    __shared__ uint32_t sharedEqual;

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        sharedEqual = 1; // Initialize shared variable
    }
    __syncthreads();

    if (x < width && y < height) {
        // Directly compute and compare within bounds
        atomicAnd(&sharedEqual, static_cast<uint32_t>(in1[y * rowSizeIn1 + x] == in2[y * rowSizeIn2 + x]));
    }

    __syncthreads();
    
    // Write back result from shared memory to global memory
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        atomicAnd(isEqual, sharedEqual);
    }
}