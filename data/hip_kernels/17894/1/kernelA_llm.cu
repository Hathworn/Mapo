#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized version of kernelA using grid-stride loop
__global__ void kernelA(float* r, float* x, float* y, float* z, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use grid-stride loop for improved parallelization
    for (int i = idx; i < size; i += stride) {
        r[i] = x[i] * y[i] + z[i];
    }
}