#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    if (i < size) {  // Conditional to prevent out-of-bounds access
        r[i] = x[i] * y[i] + z[i];
    }
}