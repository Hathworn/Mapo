#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Avoid out-of-bounds access
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}