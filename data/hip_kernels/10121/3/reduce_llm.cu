#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce(int *g_idata, int searchedNumber, int *ok) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize by removing unnecessary synchronization
    if (g_idata[i] == searchedNumber) {
        printf("Found %d on %d position %d", searchedNumber, i, *ok);
        *ok = i; // This assignment may cause race conditions if multiple threads find 'searchedNumber'
    }
}