#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

#define MEDIAN_DIMENSION  3
#define MEDIAN_LENGTH 9

#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 16

__global__ void MedianFilter_gpu(unsigned short *Device_ImageData, int Image_Width, int Image_Height) {
    __shared__ unsigned short surround[BLOCK_WIDTH*BLOCK_HEIGHT][MEDIAN_LENGTH];
    
    int iterator;
    const int Half_Of_MEDIAN_LENGTH = (MEDIAN_LENGTH / 2) + 1;
    int StartPoint = MEDIAN_DIMENSION / 2;
    int EndPoint = StartPoint + 1;

    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    const int tid = threadIdx.y * blockDim.x + threadIdx.x; // Corrected tid calculation

    if (x >= Image_Width || y >= Image_Height)
        return;

    // Fill surround with pixel value of Image in MEDIAN_DIMENSION x MEDIAN_DIMENSION pattern
    if (x >= StartPoint && x < Image_Width - StartPoint && y >= StartPoint && y < Image_Height - StartPoint) {
        iterator = 0;
        for (int r = x - StartPoint; r < x + EndPoint; r++) {
            for (int c = y - StartPoint; c < y + EndPoint; c++) {
                surround[tid][iterator] = *(Device_ImageData + (c * Image_Width) + r);
                iterator++;
            }
        }

        // Optimized sort using selection sort
        for (int i = 0; i < Half_Of_MEDIAN_LENGTH; ++i) {
            int min = i;
            for (int l = i + 1; l < MEDIAN_LENGTH; ++l) {
                if (surround[tid][l] < surround[tid][min]) {
                    min = l;
                }
            }
            if (min != i) {
                unsigned short temp = surround[tid][i];
                surround[tid][i] = surround[tid][min];
                surround[tid][min] = temp;
            }
        }

        *(Device_ImageData + (y * Image_Width) + x) = surround[tid][Half_Of_MEDIAN_LENGTH - 1];
        __syncthreads();
    }
}