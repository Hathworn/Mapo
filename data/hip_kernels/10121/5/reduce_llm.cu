#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce(int *g_idata, int *g_odata) {
    __shared__ int sdata[256];

    // Load data into shared memory with bounds check
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < 256) {
        sdata[threadIdx.x] = g_idata[i];
    } else {
        sdata[threadIdx.x] = 0;
    }
    
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) { // Reduce total iterations
        if (threadIdx.x < s) {
            sdata[threadIdx.x] += sdata[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        atomicAdd(g_odata, sdata[0]);
    }
}