#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void p2pPingPongLatencyTest(void *_pLocal, void *_pRemote, uint64_t *pTimestamps, int bWait, int cIterations)
{
    // Use volatile pointers for shared data access
    volatile int *pLocal = (volatile int *)_pLocal;
    volatile int *pRemote = (volatile int *)_pRemote;
    int pingpongValue = 0;

    // Kernel thread index
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Ensure only one thread performs the operations to prevent race conditions
    if (idx == 0) {
        while (cIterations--) {
            *pTimestamps++ = clock64(); // Record the clock at each iteration
            if (bWait) {
                // Spin-wait for the local value to match ping-pong value
                while (*pLocal != pingpongValue);
            }
            bWait = 1; // Ensure waiting in subsequent iterations
            pingpongValue = 1 - pingpongValue; // Toggle ping-pong value
            *pRemote = pingpongValue; // Update the remote value
        }
    }
}