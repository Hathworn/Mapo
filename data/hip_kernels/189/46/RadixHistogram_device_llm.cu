#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RadixHistogram_device(int *dptrHistogram, const int *in, size_t N, int shift, int mask) {
    // Calculate the global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a loop stride pattern for better coalescing and to ensure all elements are processed
    for (int i = idx; i < N; i += blockDim.x * gridDim.x) {
        int index = (in[i] & mask) >> shift;
        atomicAdd(dptrHistogram + index, 1); // Atomically add to histogram
    }
}