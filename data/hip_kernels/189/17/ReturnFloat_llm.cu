#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReturnFloat( float *sum, float *out, const float *pIn )
{
    extern __shared__ float s[];

    // Load input data into shared memory
    unsigned int tid = threadIdx.x;
    s[tid] = pIn[tid];
    __syncthreads();

    // Perform atomic addition
    atomicAdd(sum, s[tid]);

    // Write the result to the output
    out[tid] = s[tid];
}