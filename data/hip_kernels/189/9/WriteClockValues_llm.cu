#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void WriteClockValues(unsigned int *completionTimes, unsigned int *threadIDs) {
    // Simplified calculation of unique global index for each thread
    size_t globalThreadIndex = blockIdx.z * gridDim.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x +
                               blockIdx.y * gridDim.x * blockDim.z * blockDim.y * blockDim.x +
                               blockIdx.x * blockDim.z * blockDim.y * blockDim.x +
                               threadIdx.z * blockDim.y * blockDim.x +
                               threadIdx.y * blockDim.x +
                               threadIdx.x;

    // Record clock value and thread ID
    completionTimes[globalThreadIndex] = clock();
    threadIDs[globalThreadIndex] = (threadIdx.y << 4) | threadIdx.x;
}