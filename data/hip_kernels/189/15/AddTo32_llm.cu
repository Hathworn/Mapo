#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AddTo32(int *sum, int *out, const int *pIn)
{
    // Use shared memory to reduce global memory transactions
    __shared__ int sharedIn;
    
    // Load input into shared memory once per block
    if (threadIdx.x == 0)
    {
        sharedIn = *pIn;
    }
    __syncthreads(); // Ensure sharedIn is loaded before use

    // Perform atomic add using shared memory value
    atomicAdd(&out[threadIdx.x], sharedIn);
}