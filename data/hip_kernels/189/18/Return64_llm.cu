#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Return64( unsigned long long *sum, unsigned long long *out, const unsigned long long *pIn )
{
    // Use blockIdx.x to handle more than one block
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform atomic addition safely across various threads within a block
    out[index] = atomicAdd(&sum[index], *pIn);
}