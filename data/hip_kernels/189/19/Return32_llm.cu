#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Return32( int *sum, int *out, const int *pIn )
{
    // Use shared memory to cache input values
    extern __shared__ int s[];
    int tid = threadIdx.x;
    s[tid] = pIn[tid];
    __syncthreads();

    // Atomic operation only needed if modifying shared data amongst threads
    atomicAdd(&s[tid], pIn[tid]);
    __syncthreads();

    // Directly store the result in output array
    out[tid] = s[tid];
}