#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Return32( int *sum, int *out, const int *pIn )
{
    // Load input value into register to reduce memory latency
    int inputVal = *pIn;

    // Perform atomic addition for concurrent write
    out[threadIdx.x] = atomicAdd(&sum[threadIdx.x], inputVal);
}