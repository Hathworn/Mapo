#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AddToFloat(float *sum, float *out, const float *pIn)
{
    // Calculate global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use the global ID for atomic addition
    if (idx < blockDim.x) // Ensure within bounds
    {
        atomicAdd(&out[idx], pIn[idx]);
    }
}