#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReturnFloat(float *sum, float *out, const float *pIn)
{
    // Use block and thread indices for optimizing memory access
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check for valid index within the input array size for safe memory access
    if (idx < blockDim.x)
    {
        atomicAdd(&out[idx], pIn[idx]);
    }
}