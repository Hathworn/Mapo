#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void double_value(double *x, double *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global index
    y[idx] = 2. * x[idx]; // Perform value doubling
}