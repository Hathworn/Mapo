#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void checkIndex(void) {
    // Simplified printf by grouping the kernel attributes into a single statement
    printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) "
           "blockDim:(%d, %d, %d) gridDim:(%d, %d, %d)\n", 
           threadIdx.x, threadIdx.y, threadIdx.z,
           blockIdx.x, blockIdx.y, blockIdx.z, 
           blockDim.x, blockDim.y, blockDim.z,
           gridDim.x, gridDim.y, gridDim.z);
}