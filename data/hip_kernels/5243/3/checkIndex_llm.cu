#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for checking thread and block indices
__global__ void checkIndex(void) {
    // Compute global thread index for better dimensional check
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;

    // Only thread with a specific globalId performs the print operation
    if(globalId == 0) {
        printf("threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) "
               "gridDim:(%d, %d, %d)\n", threadIdx.x, threadIdx.y, threadIdx.z,
               blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z,
               gridDim.x, gridDim.y, gridDim.z);
    }
}