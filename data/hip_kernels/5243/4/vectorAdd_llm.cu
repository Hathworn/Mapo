#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(int *A, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Unrolling the loop to improve performance
    int stride = gridDim.x * blockDim.x;
    for (; i < numElements; i += stride)
    {
        A[i] = A[i] * 2;
    }
}