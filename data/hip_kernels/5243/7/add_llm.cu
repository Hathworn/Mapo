#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *fData, int *sData, int *oData, int x, int y) {

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    // Calculate total size only once
    int totalSize = x * y;

    // Optimize loop by checking condition only once per stride
    while (index < totalSize) {
        oData[index] = fData[index] + sData[index];
        index += stride;
    }
}