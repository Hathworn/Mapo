#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sub(int *fData, int *sData, int *oData, int x, int y) {

    // Calculate the global index
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use the calculated index to process data directly
    if (index < x * y) {
        oData[index] = fData[index] - sData[index];
    }
}