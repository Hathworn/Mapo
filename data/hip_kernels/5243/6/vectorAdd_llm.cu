#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(int *A, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Unroll loops to improve performance
    #pragma unroll
    for (int idx = i; idx < numElements; idx += blockDim.x * gridDim.x)
    {
        A[idx] = A[idx] * 2;
    }
}