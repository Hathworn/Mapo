#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global index based on block and thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by using shared memory for better memory access pattern
    __shared__ float shared_a[256];
    __shared__ float shared_b[256];

    const unsigned int threadLane = threadIdx.x;

    if (idx < numEls) {
        // Load data into shared memory
        shared_a[threadLane] = a[idx];
        shared_b[threadLane] = b[idx];
        __syncthreads();

        // Perform computation using shared memory
        dest[idx] = shared_a[threadLane] + shared_b[threadLane];
    }
}