#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"




hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


const int arraySize = 10000000;

__global__ void addKernel(int *c, const int *a, const int *b)
{
int i = threadIdx.x;
c[i] = a[i] + b[i];
}