#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

const int arraySize = 10000000;

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Optimize to handle full range of indices
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < arraySize) {
        c[i] = a[i] + b[i];
    }
}