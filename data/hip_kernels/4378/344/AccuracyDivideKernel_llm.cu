#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Fetch accuracy into a register to reduce global memory access
    float acc = *accuracy;
    
    // Perform division using fdividef for better performance
    acc = fdividef(acc, N);
    
    // Write the result back to global memory
    *accuracy = acc;
}