#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_regularize_kernel(int factors, float regularization, float * YtY) {
    int idx = threadIdx.x;
    // Coalesced memory access for better performance
    YtY[idx * factors + idx] += regularization; 
}