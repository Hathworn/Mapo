#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiply(float *dest, float *a, float *b, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if (i < n) // Avoid out-of-bounds memory access
    {
        dest[i] = a[i] * b[i];
    }
}