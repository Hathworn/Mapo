#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel5(int *a, int dimx, int dimy) 
{
    // Calculate flattened global index for the 2D grid
    int idx = blockIdx.y * blockDim.y * dimx + blockIdx.x * blockDim.x + threadIdx.y * dimx + threadIdx.x;

    // Check boundaries and assign value
    if (threadIdx.x < dimx && threadIdx.y < dimy) 
    {
        a[idx] = blockIdx.y;
    }
}