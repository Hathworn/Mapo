#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void k0(float* g_dataA, float* g_dataB, int pitch, int width)
{
    // global thread(data) row index
    unsigned int i = blockIdx.y * blockDim.y + threadIdx.y + 1;

    // global thread(data) column index
    unsigned int j = blockIdx.x * blockDim.x + threadIdx.x + 1;

    // check the boundary
    if (i >= width - 1 || j >= width - 1 || i < 1 || j < 1) return;

    // Pre-calculate offsets to improve access efficiency
    unsigned int idx = i * pitch + j;
    unsigned int idxN = (i - 1) * pitch + j;
    unsigned int idxS = (i + 1) * pitch + j;
    unsigned int idxE = idx + 1;
    unsigned int idxW = idx - 1;
    unsigned int idxNE = idxN + 1;
    unsigned int idxNW = idxN - 1;
    unsigned int idxSE = idxS + 1;
    unsigned int idxSW = idxS - 1;

    // Perform calculation with pre-computed indices
    g_dataB[idx] = (
        0.2f * g_dataA[idx] +
        0.1f * g_dataA[idxN] +
        0.1f * g_dataA[idxNE] +
        0.1f * g_dataA[idxE] +
        0.1f * g_dataA[idxSE] +
        0.1f * g_dataA[idxS] +
        0.1f * g_dataA[idxSW] +
        0.1f * g_dataA[idxW] +
        0.1f * g_dataA[idxNW]
    ) * 0.95f;
}