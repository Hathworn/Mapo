#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel4(int *a, int dimx, int dimy)
{
    // Calculate flattened global thread index
    int idx = blockIdx.y * blockDim.y * dimx + blockIdx.x * blockDim.x + threadIdx.y * dimx + threadIdx.x;

    // Check bounds and update the array
    if (blockIdx.x * blockDim.x + threadIdx.x < dimx && blockIdx.y * blockDim.y + threadIdx.y < dimy)
        a[idx] = (threadIdx.y * blockDim.x) + threadIdx.x;
}