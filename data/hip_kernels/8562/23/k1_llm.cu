#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void k1(float* g_dataA, float* g_dataB, int floatpitch, int width) {
    extern __shared__ float s_data[];
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    unsigned int y = blockIdx.y * blockDim.y + ty + 1;
    unsigned int x = blockIdx.x * blockDim.x + tx + 1;
    
    if (y >= width - 1 || x >= width - 1 || y < 1 || x < 1) return;

    int startCol = blockDim.x * blockIdx.x;
    int s_rowwidth = blockDim.x + 2;

    // Load shared memory inline
    #define LOAD_SHARED(idx, global_idx) s_data[idx] = g_dataA[global_idx]

    int mid_row = blockIdx.x * blockDim.x + 1 + floatpitch * blockIdx.y;
    int g_index0 = (mid_row - 1) * floatpitch + startCol + 1 + tx;
    int g_index1 = mid_row * floatpitch + startCol + 1 + tx;
    int g_index2 = (mid_row + 1) * floatpitch + startCol + 1 + tx;
    
    // Load shared memory with overlap
    LOAD_SHARED(tx + 1, g_index0);
    LOAD_SHARED(tx + s_rowwidth + 1, g_index1);
    LOAD_SHARED(tx + 2 * s_rowwidth + 1, g_index2);
    
    if (ty == 0) {
        LOAD_SHARED(tx, g_index0 - 1);
        LOAD_SHARED(tx + s_rowwidth, g_index1 - 1);
        LOAD_SHARED(tx + 2 * s_rowwidth, g_index2 - 1);
    }
    if (tx == blockDim.x - 1) {
        LOAD_SHARED(tx + 2, g_index0 + 1);
        LOAD_SHARED(tx + s_rowwidth + 2, g_index1 + 1);
        LOAD_SHARED(tx + 2 * s_rowwidth + 2, g_index2 + 1);
    }

    __syncthreads();

    // Compute the result for g_dataB
    int s_index0 = tx + 1;
    int s_index1 = s_index0 + s_rowwidth;
    int s_index2 = s_index1 + s_rowwidth;
    
    g_dataB[y * width + x] = (
        0.2f * s_data[s_index1] +
        0.1f * s_data[s_index0 - 1] +
        0.1f * s_data[s_index0 + 1] +
        0.1f * s_data[s_index1 + 1] +
        0.1f * s_data[s_index1 - 1] +
        0.1f * s_data[s_index2] +
        0.1f * s_data[s_index2 - 1] +
        0.1f * s_data[s_index2 + 1]
    ) * 0.95f;
}