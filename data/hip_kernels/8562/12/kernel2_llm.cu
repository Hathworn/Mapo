#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel function
__global__ void kernel2( int *a, int dimx, int dimy )
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensuring the thread is within bounds
    if (ix < dimx && iy < dimy) {
        int idx = iy * dimx + ix;
        a[idx] = (blockIdx.x + blockIdx.y);
    }
}