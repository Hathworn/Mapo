#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize(int *values, int *max, float *output, int n)
{
    // Calculate index based on thread and block IDs
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce access latency to 'max'
    __shared__ int shared_max;
    if (threadIdx.x == 0) {
        shared_max = *max;
    }
    __syncthreads();

    // Perform normalization only for valid indices
    if (i < n) {
        output[i] = (float)values[i] / (float)shared_max;
    }
}