#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel3( int *a, int dimx, int dimy )
{
    // Calculate the flattened thread id
    int idx = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * dimx;

    // Guard against out-of-bounds access
    if (idx < dimx * dimy) {
        a[idx] = idx;
    }
}