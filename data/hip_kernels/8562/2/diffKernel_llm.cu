#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void diffKernel(float *in, float *out, int n)
{
    // Calculate global thread ID, use blockDim.x for efficient calculation
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use conditional statement to ensure out-of-bound access is prevented
    if (id < n - 1) 
    {
        // Perform the difference operation
        out[id] = in[id + 1] - in[id];
    }
}