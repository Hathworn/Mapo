#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel6( int *a, int dimx, int dimy )
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if(ix < dimx && iy < dimy) // Check bounds before calculating index
    {
        int idx = iy * dimx + ix;
        a[idx] = blockIdx.x;
    }
}