#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce3(float *in, float *out, int size)
{
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input data into shared memory, using conditional checks
    if (index < size) {
        sdata[tid] = in[index];
    } else {
        sdata[tid] = 0;
    }
    
    __syncthreads();

    // Iterate over the shared memory for reduction with stride-based loop.
    // Using unrolling and conditional checks to improve performance
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        out[blockIdx.x] = sdata[0];
    }
}