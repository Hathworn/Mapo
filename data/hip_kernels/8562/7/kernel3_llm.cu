#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel3(int *a, int dimx, int dimy)
{
    // Calculate the flattened global index
    int idx = blockIdx.y * blockDim.y * dimx + blockIdx.x * blockDim.x + threadIdx.y * dimx + threadIdx.x;
    
    // Check bounds and prevent buffer overflow
    if (idx < dimx * dimy)
        a[idx] = idx;
}