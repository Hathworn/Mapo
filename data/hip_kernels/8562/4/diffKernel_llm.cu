#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diffKernel( float *in, float *out, int n )
{
    // Calculate the thread's unique index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure id is within bounds and apply diff operation
    if(id < n - 1) {
        out[id] = in[id + 1] - in[id];
    }
}