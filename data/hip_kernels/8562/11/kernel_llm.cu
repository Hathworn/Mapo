#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(int *a, int dimx, int dimy)
{
    // Calculate global thread index
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Bounds check to ensure index is within matrix dimensions
    if (ix < dimx && iy < dimy) {
        int idx = iy * dimx + ix;
        a[idx] = a[idx] + 1;
    }
}