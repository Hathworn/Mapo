#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(int *a, int dimx, int dimy)
{
    // Calculate the global thread ID
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = iy * dimx + ix;

    // Boundary check to ensure safe memory access
    if (ix < dimx && iy < dimy) {
        a[idx] = a[idx] + 1;
    }
}