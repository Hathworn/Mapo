#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce2(float *in, float *out, int n)
{
    extern __shared__ float sdata[];

    // load shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = (i < n) ? in[i] : 0;

    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) // use bitwise operations for efficient division
    {
        if (tid < s) // only half the threads need to read and write
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) out[blockIdx.x] = sdata[0];
}