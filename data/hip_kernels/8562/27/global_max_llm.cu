#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_max(int *values, int *max, int *reg_maxes, int num_regions, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        int region = i % num_regions;
        int val = values[i];
        // Use atomic operation only once for performance
        int old_max = atomicMax(&reg_maxes[region], val);
        if (val > old_max)
        {
            atomicMax(max, val);
        }
    }
}