#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_optimized(float *a, float *b, float *ab, size_t width)
{
    // Calculate row and column index of the matrix element
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the calculated indices are valid
    if (row < width && col < width)
    {
        // Initialize the accumulation variable
        float pvalue = 0;

        // Iterate over elements to compute the dot product
        for (int k = 0; k < width; k++)
        {
            pvalue += a[row * width + k] * b[k * width + col];
        }

        // Store the result in the output matrix
        ab[row * width + col] = pvalue;
    }
}