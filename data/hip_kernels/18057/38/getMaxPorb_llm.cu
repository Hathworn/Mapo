#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getMaxPorb(const int size, const float* class_prob, const int class_num, float* max_prob, int* idx, int *class_idx, const int conf_thresh)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread index is within bounds
    if (index >= size) return;

    // Compute memory offset once
    const float *start = class_prob + index * class_num;

    // Initialize max probability and class index
    float temp_max_prob = start[0];
    int class_index = 0;

    // Find max class probability and index
    for (int i = 1; i < class_num; i++)
    {
        float curr_prob = start[i];
        if (temp_max_prob < curr_prob)
        {
            class_index = i;
            temp_max_prob = curr_prob;
        }
    }

    // Store results with conditional threshold check
    if (temp_max_prob >= conf_thresh)
    {
        max_prob[index] = temp_max_prob;
    }
    else
    {
        max_prob[index] = 0.0f;
    }

    idx[index] = index;
    class_idx[index] = class_index;
}