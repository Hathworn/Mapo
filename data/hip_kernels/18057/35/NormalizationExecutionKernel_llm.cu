#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizationExecutionKernel(unsigned char* src, float* dst, const int size, const float alpha, const float beta, const float bias)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll the loop to handle more elements per thread where applicable
    int stride = blockDim.x * gridDim.x;
    for(; index < size; index += stride) {
        dst[index] = (static_cast<float>(src[index]) - alpha) / beta + bias; // Use static_cast for better type conversion
    }
}