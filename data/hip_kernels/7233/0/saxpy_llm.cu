#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy(int * a, int * b, int * c)
{
    // Calculate the global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop to allow for more flexible grid size
    int stride = blockDim.x * gridDim.x;

    // Optimize with loop unrolling for better performance
    for(int i = tid; i < N; i += stride * 4) {
        c[i] = 2 * a[i] + b[i];
        if (i + stride < N) c[i + stride] = 2 * a[i + stride] + b[i + stride];
        if (i + 2 * stride < N) c[i + 2 * stride] = 2 * a[i + 2 * stride] + b[i + 2 * stride];
        if (i + 3 * stride < N) c[i + 3 * stride] = 2 * a[i + 3 * stride] + b[i + 3 * stride];
    }
}