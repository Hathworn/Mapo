#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use __restrict__ to potentially improve memory access optimization
__global__ void Add(const float* __restrict__ d_a, const float* __restrict__ d_b, float* __restrict__ d_c, int N)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID
    if(id < N) // Ensure in-bounds access
    {
        d_c[id] = d_a[id] + d_b[id]; // Perform addition
    }
}