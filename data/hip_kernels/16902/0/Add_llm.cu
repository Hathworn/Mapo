#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

// Optimized GPU Code
__global__ void Add(float* d_a, float* d_b, float* d_c, int N)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds and perform operation
    if (id < N) {
        d_c[id] = d_a[id] + d_b[id];
    }
}