#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_Lbl(float* lbl, float* flo, float* cap, const int size)
{
    int idx = CUDASTDOFFSET;
    // Ensure index is within bounds before executing logic
    if (idx < size)
    {
        float value1 = cap[idx];
        float value2 = flo[idx];
        lbl[idx] = (value2 == value1) ? 1.0f : 0.0f; // Simplified conditional assignment
    }
}