#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_DivideBuffers(float* dst, float* src, const int size)
{
    int idx = CUDASTDOFFSET;

    // Only process if index is within bounds
    if (idx < size)
    {
        float value1 = src[idx];
        float value2 = dst[idx];

        // Prevent division by zero
        if (value1 != 0)
        {
            dst[idx] = value2 / value1;
        }
    }
}