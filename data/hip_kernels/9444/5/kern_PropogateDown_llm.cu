#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_PropogateDown(float* working, int span, int imageSize)
{
    int idx = CUDASTDOFFSET;
    // Check bounds before memory access to avoid redundant indexing
    if (idx + span < imageSize)
    {
        float inputValue1 = working[idx];
        float inputValue2 = working[idx + span];
        // Minimize directly to avoid additional branching
        working[idx] = fminf(inputValue1, inputValue2);
    }
}