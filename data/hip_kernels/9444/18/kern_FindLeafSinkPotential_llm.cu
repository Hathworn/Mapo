#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kern_FindLeafSinkPotential(float* sinkBuffer, float* incBuffer, float* divBuffer, float* labelBuffer, float iCC, int size)
{
    int idx = CUDASTDOFFSET;
    // Check if idx is within bounds before accessing arrays to avoid redundant computation
    if (idx < size)
    {
        float value = incBuffer[idx] - divBuffer[idx] + labelBuffer[idx] * iCC;
        sinkBuffer[idx] = value;
    }
}