#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kern_ProbBuffer(float* agreement, float* output, int size, short max)
{
    int idx = CUDASTDOFFSET;
    if (idx < size)  // Early exit for threads not needed
    {
        float locAgreement = agreement[idx];
        float probValue = fminf(locAgreement / (float)max, 1.0f);  // Use fminf for clamping
        output[idx] = probValue;
    }
}