#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define CUDASTDOFFSET (blockIdx.x * blockDim.x + threadIdx.x)

__global__ void kern_FindSourcePotentialAndStore(float* workingBuffer, float* sinkBuffer, float* divBuffer, float* labelBuffer, float iCC, int size)
{
    int idx = CUDASTDOFFSET;
  
    // Ensure index is within bounds before accessing memory
    if (idx < size) 
    {
        float value = workingBuffer[idx] + sinkBuffer[idx] + divBuffer[idx] - labelBuffer[idx] * iCC;
        workingBuffer[idx] = value;
    }
}