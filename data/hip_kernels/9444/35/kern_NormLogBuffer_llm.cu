#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kern_NormLogBuffer(float* agreement, float* output, float maxOut, int size, short max)
{
    int idx = CUDASTDOFFSET;
    if (idx < size) {  // Ensure idx is within bounds
        float locAgreement = agreement[idx];
        float logValue = (locAgreement > 0.0f) ? log((float)max) - log(locAgreement) : maxOut;

        // Bound logValue within [0, maxOut] and scale
        if (logValue < 0.0f) logValue = 0.0f;
        else if (logValue > maxOut) logValue = 1.0f;
        else logValue /= maxOut;

        output[idx] = logValue;
    }
}