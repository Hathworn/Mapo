#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_FindSinkPotentialAndStore(float* workingBuffer, float* incBuffer, float* divBuffer, float* labelBuffer, float iCC, int size)
{
    int idx = CUDASTDOFFSET;

    // Ensure idx is within bounds before accessing arrays
    if (idx < size)
    {
        float incVal = incBuffer[idx];       // Load from incBuffer
        float divVal = divBuffer[idx];       // Load from divBuffer
        float labelVal = labelBuffer[idx];   // Load from labelBuffer

        // Compute value with reduced memory access
        float value = workingBuffer[idx] + incVal - divVal + labelVal * iCC;

        // Store computed value back into workingBuffer
        workingBuffer[idx] = value;
    }
}