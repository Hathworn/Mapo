#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CUDAkernel_multiply(float* sourceA, float* sourceB, float* destination, int size)
{
    int index = CUDASTDOFFSET;

    // Load data only if within bounds
    if (index < size)
    {
        float a = sourceA[index];
        float b = sourceB[index];
        destination[index] = a * b;
    }
}