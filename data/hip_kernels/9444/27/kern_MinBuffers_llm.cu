#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_MinBuffers(float* b1, float* b2, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index
    
    if(idx < size) // Perform operations only if index is within size
    {
        float value1 = b1[idx];
        float value2 = b2[idx];
        b1[idx] = (value1 < value2) ? value1 : value2; // Directly assign min value to b1
    }
}