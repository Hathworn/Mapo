#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_ConvertBuffer(short* agreement, float* output, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Use HIP specific index calculation

    if (idx < size)
    {
        output[idx] = static_cast<float>(agreement[idx]); // Use static_cast for type conversion
    }
}