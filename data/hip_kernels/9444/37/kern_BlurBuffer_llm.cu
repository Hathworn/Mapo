#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kern_BlurBuffer(float* input, float* output, int size, int spread, int dim)
{
    int idx = CUDASTDOFFSET;
    if (idx < size) { // Ensure index is within bounds
        int x = (idx / spread) % dim;
        float curr = input[idx];
        float down = (idx - spread >= 0) ? input[idx - spread] : curr; // Use curr if out of bounds
        float up = (idx + spread < size) ? input[idx + spread] : curr; // Use curr if out of bounds
        float newVal = 0.7865707f * curr + 0.1064508f * (down + up);
        output[idx] = newVal;
    }
}