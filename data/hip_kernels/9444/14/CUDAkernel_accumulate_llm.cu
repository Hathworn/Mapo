#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function with thread bounds checking
__global__ void CUDAkernel_accumulate(float* buffer, int addSize, int size)
{
    int index = CUDASTDOFFSET;

    // Ensure index is within buffer bounds
    if (index < size && index + addSize < size) 
    {
        float a = buffer[index];
        float b = buffer[index + addSize];
        
        buffer[index] = a + b; // Perform accumulation
    }
}