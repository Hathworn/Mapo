#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kern_Copy2Buffers(float* fIn, float* fOut1, float* fOut2, int size)
{
    int idx = CUDASTDOFFSET;
    // Check boundary condition once and compute memory address
    if( idx < size )
    {
        float value = fIn[idx];
        fOut1[idx] = value;
        fOut2[idx] = value;
    }
}