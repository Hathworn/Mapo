#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kern_PropogateUp(float* working, int span, int imageSize)
{
    int idx = CUDASTDOFFSET;

    // Ensure idx+span is within bounds first to avoid unnecessary work
    if (idx + span < imageSize)
    {
        // Initialize shared variables for better memory access
        float inputValue1 = working[idx];
        float inputValue2 = working[idx + span];
        
        // Calculate output value using conditional operator
        float outputVal = (inputValue1 > inputValue2) ? inputValue1 : inputValue2;

        // Update working array
        working[idx] = outputVal;
    }
}