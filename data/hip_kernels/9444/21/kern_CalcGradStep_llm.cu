#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_CalcGradStep(float* sinkBuffer, float* incBuffer, float* divBuffer, float* labelBuffer, float stepSize, float iCC, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index is within the bounds
    if( idx < size )
    {
        // Improved calculation to avoid redundant operations
        float value = stepSize * (sinkBuffer[idx] + divBuffer[idx] - incBuffer[idx] - labelBuffer[idx] * iCC);
        divBuffer[idx] = value;
    }
}