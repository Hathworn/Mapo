#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinSample(float* DistanceBuffer, short2* IndexBuffer, int spread, int mapSizeX, int mapSizeY)
{
    int kOffset = CUDASTDOFFSET;
    
    // Pre-check index bounds for better performance
    if (kOffset + spread < mapSizeX * mapSizeY)
    {
        float distance1 = DistanceBuffer[kOffset];
        float distance2 = DistanceBuffer[kOffset + spread];
        short2 index1 = IndexBuffer[kOffset];
        short2 index2 = IndexBuffer[kOffset + spread];
        
        // Directly compute without temporary storage
        bool isDistance1Smaller = distance1 < distance2;
        DistanceBuffer[kOffset] = isDistance1Smaller ? distance1 : distance2;
        IndexBuffer[kOffset] = isDistance1Smaller ? index1 : index2;
    }
}