#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SetBufferToRandom(float* buffer, float min, float max, int size){
int offset = CUDASTDOFFSET;
hiprandState localState;
hiprand_init(7+offset, offset, 0, &localState);
__syncthreads();

float value = min + (max-min)*hiprand_uniform(&localState);
if(offset < size ) buffer[offset] = value;
}