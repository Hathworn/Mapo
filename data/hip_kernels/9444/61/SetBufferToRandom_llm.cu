#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SetBufferToRandom(float* buffer, float min, float max, int size) {
    int offset = CUDASTDOFFSET;

    // Initialize local state for the current thread
    hiprandState localState;
    hiprand_init(7 + offset, offset, 0, &localState);

    // Ensure all threads have initialized their state before proceeding
    __syncthreads();

    // Check if offset is within bounds before computing random value
    if (offset < size) {
        // Generate random value only for valid offsets
        float value = min + (max - min) * hiprand_uniform(&localState);
        buffer[offset] = value;
    }
}