#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SumOverLargeBuffer(float* buffer, int spread, int size) {
    int offset = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (offset + spread < size) { // Ensure within bounds
        buffer[offset] += buffer[offset + spread]; // Direct in-place addition
    }
}