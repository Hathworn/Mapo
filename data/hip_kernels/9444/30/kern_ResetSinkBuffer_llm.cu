#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kern_ResetSinkBuffer(float* sink, float* source, float* div, float* label, float ik, float iCC, int size)
{
    int idx = CUDASTDOFFSET;
    if (idx < size)
    {
        // Consolidated calculation into single expression
        sink[idx] = (1.0f - ik) * sink[idx] + ik * (source[idx] - div[idx] + label[idx] * iCC);
    }
}