#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReplaceNANs(float* buffer, float value, int size) {
    int offset = CUDASTDOFFSET;
    if (offset < size) {
        float current = buffer[offset];
        // Check if the current value is finite and replace if necessary
        buffer[offset] = isfinite(current) ? current : value;
    }
}