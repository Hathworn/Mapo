#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kern_ApplyCapacity(float* sinkBuffer, float* capBuffer, int size)
{
    int idx = CUDASTDOFFSET;

    // Ensure idx is within bounds before all operations to avoid redundant checks
    if (idx < size)
    {
        float value = sinkBuffer[idx];
        float cap = capBuffer[idx];
        
        // Utilize fmaxf and fminf for cleaner clamping
        value = fmaxf(0.0f, fminf(value, cap));
        
        sinkBuffer[idx] = value;
    }
}