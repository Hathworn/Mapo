#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void hello()
{
    // Each thread will print its thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    printf("Hello world! I'm thread %d\n", idx); // Use global index for printing
}