#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
    // Perform warp-level reduction using shfl_down to minimize operations
    unsigned int lane = threadIdx.x % warpSize;
    s[lane] = p;
    for (int offset = 1; offset < warpSize; offset *= 2) {
        unsigned int val = __shfl_down(p, offset);
        p += val;
    }
    return p;
}

__global__ void reduce(unsigned int * d_out_shared, const unsigned int * d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    
    // Store the final reduction result in d_out_shared
    if (t % warpSize == 0) {
        atomicAdd(d_out_shared, sr);
    }
}