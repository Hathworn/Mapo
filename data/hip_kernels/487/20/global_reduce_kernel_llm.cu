#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_reduce_kernel(float *d_out, float *d_in)
{
    extern __shared__ float s_data[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    
    // Load data into shared memory for faster access
    s_data[tid] = d_in[myId];
    __syncthreads();
    
    // Perform reduction within shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads(); // Ensure all additions at one stage are completed
    }

    // Only thread 0 writes result for this block back to global memory
    if (tid == 0)
    {
        d_out[blockIdx.x] = s_data[0];
    }
}