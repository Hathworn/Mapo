#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel function by using threadIdx and gridDim for fine-grained control
__global__ void hello()
{
    // Calculate global thread index
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Print statement using thread and block indices
    printf("Hello world! I'm thread %d in block %d\n", globalThreadId, blockIdx.x);
}