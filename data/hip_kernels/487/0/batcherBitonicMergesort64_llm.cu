#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    extern __shared__ float sdata[];
    int tid  = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    for (int stage = 0; stage <= 5; stage++)
    {
        for (int substage = stage; substage >= 0; substage--)
        {
            int pairDistance = 1 << substage;
            int blockSize = 1 << (stage + 1);
            int pairFilter = blockSize / 2;

            int sortedDirection = ((tid & pairFilter) == 0) ? 1 : -1;
            int compareIndex = tid ^ pairDistance;

            if (compareIndex > tid) 
            {
                float val1 = sdata[tid];
                float val2 = sdata[compareIndex];
                // Swap if needed
                if ((val1 > val2 && sortedDirection == 1) || (val1 < val2 && sortedDirection == -1)) 
                {
                    sdata[tid] = val2;
                    sdata[compareIndex] = val1;
                }
            }
            __syncthreads(); // Ensure all threads have written
        }
    }

    d_out[tid] = sdata[tid];
}