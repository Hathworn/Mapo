#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shmem_reduce_kernel(float * d_out, const float * d_in)
{
    // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // load shared mem from global mem
    sdata[tid] = d_in[myId];
    __syncthreads(); // ensure shared memory is fully populated

    // warp-level reduction optimization
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // sync only when necessary
    }

    // unroll the last warp
    if (tid < 32)
    {
        volatile float* v_sdata = sdata; // prevent data relocation
        v_sdata[tid] += v_sdata[tid + 32];
        v_sdata[tid] += v_sdata[tid + 16];
        v_sdata[tid] += v_sdata[tid + 8];
        v_sdata[tid] += v_sdata[tid + 4];
        v_sdata[tid] += v_sdata[tid + 2];
        v_sdata[tid] += v_sdata[tid + 1];
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}