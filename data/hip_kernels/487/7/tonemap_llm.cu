#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tonemap(float* d_x, float* d_y, float* d_log_Y, float* d_cdf_norm, float* d_r_new, float* d_g_new, float* d_b_new, float min_log_Y, float max_log_Y, float log_Y_range, int num_bins, int num_pixels_y, int num_pixels_x)
{
    int nx = num_pixels_x;
    int image_index_1d = (blockIdx.y * blockDim.y + threadIdx.y) * nx + (blockIdx.x * blockDim.x + threadIdx.x);

    if (blockIdx.x * blockDim.x + threadIdx.x < nx && blockIdx.y * blockDim.y + threadIdx.y < num_pixels_y)
    {
        float x = d_x[image_index_1d];
        float y = d_y[image_index_1d];
        float log_Y = d_log_Y[image_index_1d];
        int bin_index = min(num_bins - 1, int((num_bins * (log_Y - min_log_Y)) / log_Y_range));
        float Y_new = d_cdf_norm[bin_index];

        float X_new = x * (Y_new / y);
        float Z_new = (1 - x - y) * (Y_new / y);

        // Calculate RGB values using linear transformation
        d_r_new[image_index_1d] = (X_new * 3.2406f) + (Y_new * -1.5372f) + (Z_new * -0.4986f);
        d_g_new[image_index_1d] = (X_new * -0.9689f) + (Y_new * 1.8758f) + (Z_new * 0.0415f);
        d_b_new[image_index_1d] = (X_new * 0.0557f) + (Y_new * -0.2040f) + (Z_new * 1.0570f);
    }
}