#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth(float * v_new, const float * v) {
    // Calculate global thread index
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;
    
    // Calculate indices for neighboring elements
    int myLeftIdx = max(0, myIdx - 1);  // Prevent out-of-bounds access at start
    int myRightIdx = min(numThreads - 1, myIdx + 1);  // Prevent out-of-bounds access at end
    
    // Load element and its neighbors
    float myElt = v[myIdx];
    float myLeftElt = v[myLeftIdx];
    float myRightElt = v[myRightIdx];

    // Compute the new value with smoothing
    v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
}