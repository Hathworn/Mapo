#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void remove_redness_from_coordinates(const unsigned int* d_coordinates, unsigned char* d_r, unsigned char* d_b, unsigned char* d_g, unsigned char* d_r_output, int num_coordinates, int num_pixels_y, int num_pixels_x, int template_half_height, int template_half_width)
{
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;
    int imgSize = num_pixels_x * num_pixels_y;

    if (global_index_1d < num_coordinates)
    {
        // Avoid reverse lookup by using coherent index calculation
        unsigned int image_index_1d = d_coordinates[global_index_1d];
        ushort2 image_index_2d = make_ushort2(image_index_1d % num_pixels_x, image_index_1d / num_pixels_x);

        int yStart = max(0, image_index_2d.y - template_half_height);
        int yEnd = min(num_pixels_y - 1, image_index_2d.y + template_half_height);
        int xStart = max(0, image_index_2d.x - template_half_width);
        int xEnd = min(num_pixels_x - 1, image_index_2d.x + template_half_width);

        for (int y = yStart; y <= yEnd; y++)
        {
            for (int x = xStart; x <= xEnd; x++)
            {
                int image_offset_index_1d = y * num_pixels_x + x;

                unsigned char g_value = d_g[image_offset_index_1d];
                unsigned char b_value = d_b[image_offset_index_1d];

                // Inline calculation of gb_average
                d_r_output[image_offset_index_1d] = (g_value + b_value) / 2;
            }
        }
    }
}