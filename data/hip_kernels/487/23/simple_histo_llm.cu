#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Calculate global thread ID
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Ensure the thread processes a valid item
    if (myId < BIN_COUNT) {
        int myItem = d_in[myId];
        int myBin = myItem % BIN_COUNT;

        // Use atomic addition to avoid race conditions
        atomicAdd(&(d_bins[myBin]), 1);
    }
}