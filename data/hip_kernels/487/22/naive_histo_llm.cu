#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naive_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;
    
    // Reduction to avoid race conditions
    __shared__ int local_bins[1024]; // Adjust the size according to BIN_COUNT if needed
    local_bins[threadIdx.x] = 0;
    __syncthreads();
    
    atomicAdd(&local_bins[myBin], 1); 
    __syncthreads();
    
    // Transfer back to global memory
    atomicAdd(&d_bins[myBin], local_bins[threadIdx.x]);
}