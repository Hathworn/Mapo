#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void prepareVerticalPottsProblems8ADMM(float* in, float* u, float* v, float* w_, float* z, float* weights, float* weightsPrime, float* lam1, float* lam4, float* lam5, float mu, uint32_t w, uint32_t h, uint32_t nc) {
    uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;
    uint32_t c = threadIdx.z + blockDim.z * blockIdx.z;

    if (x < w && y < h && c < nc) {
        uint32_t index = x + w * y + w * h * c;
        uint32_t weightsIndex = x + w * y;

        float term1 = weights[weightsIndex] * in[index];
        float term2 = 2 * mu * (u[index] + w_[index] + z[index]);
        float term3 = 2 * (lam1[index] - lam4[index] - lam5[index]);

        v[index] = (term1 + term2 + term3) / weightsPrime[weightsIndex];
    }
}