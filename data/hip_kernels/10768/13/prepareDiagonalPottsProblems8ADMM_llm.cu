#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prepareDiagonalPottsProblems8ADMM(float* in, float* u, float* v, float* w_, float* z, float* weights, float* weightsPrime, float* lam2, float* lam4, float* lam6, float mu, uint32_t w, uint32_t h, uint32_t nc) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t c = blockIdx.z * blockDim.z + threadIdx.z;

    // Boundary check to ensure memory accesses are within bounds
    if (x < w && y < h && c < nc) {
        // Calculate index for accessing data
        uint32_t index = x + w * y + w * h * c;
        uint32_t weightsIndex = x + w * y;

        // Optimize computation by storing repeated calculations in variables
        float in_val = in[index];
        float sum_uvz = u[index] + v[index] + z[index];
        float sum_lam = lam2[index] + lam4[index] - lam6[index];

         // Perform computation and store result
        w_[index] = (weights[weightsIndex] * in_val + 2 * mu * sum_uvz + 2 * sum_lam) / weightsPrime[weightsIndex];
    }
}