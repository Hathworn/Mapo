#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prepareVerticalPottsProblems4ADMM(float* in, float* u, float* v, float* weights, float* weightsPrime, float* lam, float mu, uint32_t w, uint32_t h, uint32_t nc) {
    // Calculate a single unique index for the thread
    uint32_t index = (threadIdx.z + blockDim.z * blockIdx.z) * w * h + 
                     (threadIdx.y + blockDim.y * blockIdx.y) * w + 
                     (threadIdx.x + blockDim.x * blockIdx.x);

    if (index < w * h * nc) { // Overall flatten index compared to the total size
        // Calculate weightsIndex based on x, y
        uint32_t weightsIndex = index % (w * h);

        // Use the calculated index to update v
        v[index] = (weights[weightsIndex] * in[index] + u[index] * mu + lam[index]) / weightsPrime[weightsIndex];
    }
}