#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prepareHorizontalPottsProblems8ADMM(float* in, float* u, float* v, float* w_, float* z, 
                                                    float* weights, float* weightsPrime, float* lam1, 
                                                    float* lam2, float* lam3, float mu, uint32_t w, 
                                                    uint32_t h, uint32_t nc) {
    // Calculate global thread indices for x, y, c
    uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;
    uint32_t c = threadIdx.z + blockDim.z * blockIdx.z;

    // Check boundary conditions for each dimension
    if(x < w && y < h && c < nc) {
        // Calculate linear indices for accessing arrays
        uint32_t index = x + w * y + w * h * c;
        uint32_t weightsIndex = x + w * y;

        // Perform computation and update u array
        float tempSum = weights[weightsIndex] * in[index] + 2.0f * mu * (v[index] + w_[index] + z[index])
                        - 2.0f * (lam1[index] + lam2[index] + lam3[index]);  // Simplified operations
        u[index] = tempSum / weightsPrime[weightsIndex];  // Division remains unchanged
    }
}