#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prepareAntidiagonalPottsProblems8ADMM(float* in, float* u, float* v, float* w_, float* z, 
                                                      float* weights, float* weightsPrime, float* lam3, 
                                                      float* lam5, float* lam6, float mu, uint32_t w, 
                                                      uint32_t h, uint32_t nc) {
    // Cache thread ID calculations in registers for performance
    uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
    uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;
    uint32_t c = threadIdx.z + blockDim.z * blockIdx.z;

    // Ensure computations are only performed for valid indices
    if (x < w && y < h && c < nc) {
        // Precompute shared index calculations to reduce redundant computation
        uint32_t index = x + w * y + w * h * c;
        uint32_t weightsIndex = x + w * y;
        
        // Optimize calculation by reducing repeated operations
        float weightedSum = weights[weightsIndex] * in[index] + 2 * mu * (u[index] + v[index] + w_[index]) 
                            + 2 * (lam3[index] + lam5[index] + lam6[index]);

        // Use precomputed weighted sum to update z
        z[index] = weightedSum / weightsPrime[weightsIndex];
    }
}