#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateLagrangeMultiplierKernel4ADMM(float* u, float* v, float* lam, float* temp, float mu, uint32_t w, uint32_t h, uint32_t nc) {
    // Calculate index to avoid redundant computation 
    uint32_t index = threadIdx.x + blockDim.x * blockIdx.x + (threadIdx.y + blockDim.y * blockIdx.y) * w + (threadIdx.z + blockDim.z * blockIdx.z) * w * h;

    // Check if index is within bounds
    if (index < w * h * nc) {
        // Update temp and lam directly using the precalculated index
        float diff = u[index] - v[index]; // Calculate difference only once
        temp[index] = diff;
        lam[index] += diff * mu; // Compound assignment for efficiency
    }
}