#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PondHeadInit(double *ph, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (; tid < size; tid += stride) { // Use for-loop for potentially better compiler optimization
        ph[tid] = psi_min; // Set value using coalesced memory access pattern
    }
}