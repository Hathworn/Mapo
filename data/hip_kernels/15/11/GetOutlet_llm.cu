#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GetOutlet(double *h, double *houtlet, double *u, double *uout, double *v, double *vout, int M, int N, int t) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Cache the common (tid + 1) * N - ind index calculation
    int ind = 2;
    int offset = N - ind;

    while (tid < M) {
        int idx = (tid + 1) * offset;
        houtlet[t * M + tid] = h[idx];
        vout[t * M + tid] = v[idx];
        uout[t * M + tid] = u[idx];
        tid += blockDim.x * gridDim.x;
    }
}