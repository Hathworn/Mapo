#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getqss(double *IN, double *qss, int N, int t) {
    // Load indices into registers to minimize memory access
    const int i = 10; 
    const int j = 10;

    // Calculate the index using pre-computed i and j
    const int I = j * N + i;

    // Directly assign the value to qss without extra computations
    qss[t] = IN[I];
}