#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SweHInit(double *var_in1, double *var_in2, double *var_out, int size) {
    // Calculate thread id
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Optimize loop by checking boundary condition first
    if (tid < size) {
        var_out[tid] = var_in1[tid] - var_in2[tid];
        
        // Loop through remaining elements using stride to avoid excess if check
        for (int i = tid + blockDim.x * gridDim.x; i < size; i += blockDim.x * gridDim.x) {
            var_out[i] = var_in1[i] - var_in2[i];
        }
    }
}