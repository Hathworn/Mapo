#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TopForcing(double ppt, double *eff_rain, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Precompute stride to reduce redundant calculations
    for (int i = tid; i < size; i += stride) {
        eff_rain[i] = ppt; // Maintain memory coalescence by processing contiguous indices
    }
}