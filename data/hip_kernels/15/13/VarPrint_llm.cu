#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VarPrint(double *Var, int M, int N, int P) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int totalElements = M * N * P;
    
    // Ensure the thread accesses valid indices
    if (idx < totalElements) {
        int k = idx / (M * N);
        int i = (idx % (M * N)) / M;
        int j = idx % M;
        
        // Print values at calculated indices
        printf("%4.3f ", Var[k * M * N + i * M + j]);

        // Synchronize threads within the block for ordered printing
        __syncthreads();

        // Print new lines based on thread indices
        if (j == M - 1) printf("\n");
        if (i == N - 1 && j == M - 1) printf("\n\n");
    }
}