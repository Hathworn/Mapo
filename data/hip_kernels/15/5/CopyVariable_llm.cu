#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CopyVariable(double *var_in, double *var_out, int size) {
    // Calculate the global thread ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use stride loop for better occupancy
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < size; i += stride) {
        var_out[i] = var_in[i];
    }
}