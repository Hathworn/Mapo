#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LeftRightBound2D(double *Hs, double *Ztopo, double *K2e, double *K2w, int BC2D, int M, int N) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while (tid < M) {
        int baseIndex = tid * N;
        int lastIndex = (tid + 1) * N - 1;
        
        // Apply boundary conditions based on BC2D value
        if (BC2D == 0) { // No-flow BCs
            Hs[baseIndex] = Hs[baseIndex + 1];
            Hs[lastIndex] = Hs[lastIndex - 1];
        } else { // Critical depth flow BCs
            Hs[baseIndex] = hcri + Ztopo[baseIndex];
            Hs[lastIndex] = hcri + Ztopo[lastIndex];
        }
        
        // Update K2w and K2e arrays
        K2w[baseIndex] = K2w[baseIndex + 1];
        K2e[lastIndex] = K2e[lastIndex - 2];
        
        tid += stride; // Move to next element in strides
    }
}