#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE  32
#define N           3200

__global__ void matMult(float* a, float* b, int n, float* c)
{
    // Cache the sub-matrix of A and B in shared memory
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float sum = 0.0f;

    // Loop over the A and B tiles required to compute the block sub-matrix
    for (int m = 0; m < (n + BLOCK_SIZE - 1) / BLOCK_SIZE; ++m) {
        // Load A and B tiles into shared memory
        if (m * BLOCK_SIZE + tx < n && by * BLOCK_SIZE + ty < n)
            As[ty][tx] = a[(by * BLOCK_SIZE + ty) * n + m * BLOCK_SIZE + tx];
        else
            As[ty][tx] = 0.0f;

        if (m * BLOCK_SIZE + ty < n && bx * BLOCK_SIZE + tx < n)
            Bs[ty][tx] = b[(m * BLOCK_SIZE + ty) * n + bx * BLOCK_SIZE + tx];
        else
            Bs[ty][tx] = 0.0f;

        __syncthreads();

        // Multiply the two matrices together
        for (int k = 0; k < BLOCK_SIZE; ++k)
            sum += As[ty][k] * Bs[k][tx];

        __syncthreads();
    }

    // Write the block sub-matrix to device memory
    if (by * BLOCK_SIZE + ty < n && bx * BLOCK_SIZE + tx < n)
        c[(by * BLOCK_SIZE + ty) * n + bx * BLOCK_SIZE + tx] = sum;
}