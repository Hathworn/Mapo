#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Ensure thread index is within bounds
    if(i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);

        // Use ternary operator for conditional assignments
        error[i] = (abs_val < 1) ? (diff * diff) : (2 * abs_val - 1);
        delta[i] = (abs_val < 1) ? diff : ((diff > 0) ? 1.0f : -1.0f);
    }
}