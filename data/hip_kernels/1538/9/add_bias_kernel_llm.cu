#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate total number of threads
    for (int id = index; id < n * size * batch; id += stride) { // Use stride loop for full utilization
        int i = id % size;
        int j = (id / size) % n; // Combined index division
        int k = id / (size * n);

        output[(k * n + j) * size + i] += biases[j];
    }
}