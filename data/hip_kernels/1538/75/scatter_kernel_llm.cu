#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scatter_kernel( int *x_coors, int *y_coors, float *pfe_output, float *scattered_feature, const int MAX_NUM_PILLARS_, const int GRID_X_SIZE, const int GRID_Y_SIZE)
{
    // Cache the indices in shared memory to reduce global memory accesses
    extern __shared__ int shared_mem[];
    int *shared_x_coors = &shared_mem[0];
    int *shared_y_coors = &shared_mem[MAX_NUM_PILLARS_];

    int i_pillar = blockIdx.x;
    int i_feature = threadIdx.x;

    if (i_feature == 0) { 
        shared_x_coors[i_pillar] = x_coors[i_pillar];
        shared_y_coors[i_pillar] = y_coors[i_pillar];
    }
    __syncthreads();

    int x_ind = shared_x_coors[i_pillar];
    int y_ind = shared_y_coors[i_pillar];

    float feature = pfe_output[i_feature*MAX_NUM_PILLARS_ + i_pillar];
    scattered_feature[i_feature*GRID_Y_SIZE*GRID_X_SIZE + y_ind * GRID_X_SIZE + x_ind] = feature;
}