#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    // Precompute dimensions
    int h = (in_h + 2*pad) / stride;
    int w = (in_w + 2*pad) / stride;
    int c = in_c;

    // Calculate unique thread index and early exit if out of bounds
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    // Precompute offsets
    int w_offset = -pad;
    int h_offset = -pad;

    // Calculate indices
    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    // Precompute output index
    int out_index = j + w * (i + h * (k + c * b));

    // Initialize max value and index
    float max = -INFINITY;
    int max_i = -1;

    // Iterate over the pooling window
    for(int l = 0; l < size; ++l) {
        for(int m = 0; m < size; ++m) {
            int cur_h = h_offset + i * stride + l;
            int cur_w = w_offset + j * stride + m;
            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));

            // Check if current index is valid
            if (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w) {
                float val = input[index];
                if (val > max) {
                    max = val;
                    max_i = index;
                }
            }
        }
    }

    // Write results to output
    output[out_index] = max;
    indexes[out_index] = max_i;
}