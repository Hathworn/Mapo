#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate a unique global index for each thread
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Ensures the work is within bounds
    if(i < n){
        float dc_val = dc[i];
        float s_val = s[i];

        // Conditionally update da and db
        if (da) da[i] += dc_val * s_val;  // Weight update for da
        if (db) db[i] += dc_val * (1 - s_val);  // Weight update for db

        // Always update ds
        ds[i] += dc_val * (a[i] - b[i]);  // Update for ds
    }
}