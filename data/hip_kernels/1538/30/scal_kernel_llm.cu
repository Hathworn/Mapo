#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use shared memory for coalesced access
    __shared__ float shared_X[1024]; 
    
    // Calculate the global index
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    
    // Check and load element into shared memory
    if(i < N)
    {
        shared_X[threadIdx.x] = X[i * INCX];
        
        // Synchronize all threads
        __syncthreads();
        
        // Multiply the element by ALPHA
        shared_X[threadIdx.x] *= ALPHA;
        
        // Synchronize all threads
        __syncthreads();
        
        // Store results back to global memory
        X[i * INCX] = shared_X[threadIdx.x];
    }
}