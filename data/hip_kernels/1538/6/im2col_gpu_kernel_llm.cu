#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void im2col_gpu_kernel(const int n, const float* data_im, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_col) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Process multiple elements per thread to reduce total iterations
    int step = blockDim.x * gridDim.x;
    for (; index < n; index += step) {
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;

        // Pre-calculate common terms
        int h_in_origin = h_out * stride - pad;
        int w_in_origin = w_out * stride - pad;

        float* data_col_ptr = data_col + (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im + channel_in * height * width;

        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in_origin + i;
                int w = w_in_origin + j;

                // Use ternary operator to select proper value
                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                                data_im_ptr[h * width + w] : 0;

                data_col_ptr += height_col * width_col;
            }
        }
    }
}