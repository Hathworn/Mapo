#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Improve readability and performance by calculating index directly
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockDim.x * blockDim.y * blockIdx.y;
    
    // Use shared memory for better performance if required (context-dependent)
    if(i < n) {
        float si = s[i];
        float ai = a[i];
        float bi = (b ? b[i] : 0); // Utilize ternary operation for efficiency
        c[i] = si * ai + (1 - si) * bi;
    }
}