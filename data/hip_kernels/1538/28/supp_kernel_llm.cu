#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check within bounds and perform computation
    if (i < N) {
        float val = X[i * INCX];
        if (val * val < ALPHA * ALPHA) {
            X[i * INCX] = 0;
        }
    }
}