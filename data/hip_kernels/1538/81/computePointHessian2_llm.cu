#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computePointHessian2(float *x, float *y, float *z, int points_num, int *valid_points, int valid_points_num, double *dh_ang, double *ph155, double *ph165, double *ph175) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    __shared__ double h_ang[9];

    // Load shared memory efficiently
    if (threadIdx.x < 9) {
        h_ang[threadIdx.x] = dh_ang[36 + threadIdx.x];
    }

    __syncthreads();

    for (int i = id; i < valid_points_num; i += stride) {
        int pid = valid_points[i];

        // Load original coordinates
        double o_x = static_cast<double>(x[pid]);
        double o_y = static_cast<double>(y[pid]);
        double o_z = static_cast<double>(z[pid]);

        // Compute the Hessians more efficiently
        double x_h_ang1 = o_x * h_ang[1];
        double y_h_ang0 = o_y * h_ang[0];
        double z_h_ang2 = o_z * h_ang[2];
        ph155[i] = y_h_ang0 + x_h_ang1 + z_h_ang2;

        double x_h_ang4 = o_x * h_ang[4];
        double y_h_ang3 = o_y * h_ang[3];
        double z_h_ang5 = o_z * h_ang[5];
        ph165[i] = y_h_ang3 + x_h_ang4 + z_h_ang5;

        double x_h_ang7 = o_x * h_ang[7];
        double y_h_ang6 = o_y * h_ang[6];
        double z_h_ang8 = o_z * h_ang[8];
        ph175[i] = y_h_ang6 + x_h_ang7 + z_h_ang8;
    }
}