#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int c = in_c;
    int area = (size - 1) / stride;

    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplify linear index computation
    if (id >= n) return;

    int j = id % in_w;
    int i = (id / in_w) % in_h;
    int k = (id / (in_w * in_h)) % in_c;
    int b = id / (in_w * in_h * in_c);

    int w_offset = -pad;
    int h_offset = -pad;

    float d = 0;
    // Unroll loops to improve performance (e.g., through compiler optimizations)
    for (int l = -area; l <= area; ++l) {
        int out_h = (i - h_offset) / stride + l;
        if (out_h < 0 || out_h >= h) continue; // Check boundaries

        for (int m = -area; m <= area; ++m) {
            int out_w = (j - w_offset) / stride + m;
            if(out_w < 0 || out_w >= w) continue; // Check boundaries

            int out_index = out_w + w * (out_h + h * (k + c * b));
            if (indexes[out_index] == id) {
                d += delta[out_index];
            }
        }
    }
    prev_delta[id] += d;
}