#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeScoreList(int *starting_voxel_id, int *voxel_id, int valid_points_num, double *e_x_cov_x, double gauss_d1, double *score)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory for score accumulation
    extern __shared__ double shared_score[];

    for (int i = id; i < valid_points_num; i += stride) {
        shared_score[threadIdx.x] = 0;

        // Reduce branch divergence by rearranging conditions
        for (int vid = starting_voxel_id[i]; vid < starting_voxel_id[i + 1]; vid++) {
            double tmp_ex = e_x_cov_x[vid];
            if (tmp_ex <= 1 && tmp_ex >= 0 && tmp_ex == tmp_ex) {
                shared_score[threadIdx.x] += -gauss_d1 * tmp_ex;
            }
        }

        // Write result to global memory
        score[i] = shared_score[threadIdx.x];
    }
}