#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use more than one thread per block for better performance
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Calculate the stride for threads

    for (; i < N; i += stride) {
        X[i * INCX] = ALPHA; // Ensure threads cover the entire range
    }
}