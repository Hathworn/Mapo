#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate the global index for the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a stride to cover the entire range efficiently
    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x) {
        float t = truth[idx];
        float p = pred[idx];

        // Compute the error and delta for the current index
        error[idx] = -t * logf(p + 0.0000001f) - (1 - t) * logf(1 - p + 0.0000001f);
        delta[idx] = t - p;
    }
}