#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output) {
    int i;
    float sum = 0.0f;
    float largest = -INFINITY;

    // Optimize with unrolling for loop to find largest value
    for (i = 0; i < n; i += 4) {
        if (i < n) largest = fmaxf(largest, input[i * stride]);
        if (i + 1 < n) largest = fmaxf(largest, input[(i + 1) * stride]);
        if (i + 2 < n) largest = fmaxf(largest, input[(i + 2) * stride]);
        if (i + 3 < n) largest = fmaxf(largest, input[(i + 3) * stride]);
    }

    // Calculate exponentials and sum them
    for (i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp);
        sum += e;
        output[i * stride] = e;
    }

    // Normalize the output
    for (i = 0; i < n; ++i) {
        output[i * stride] /= sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset) {
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;

    int s = id % spatial;
    id /= spatial;
    int g = id % groups;
    int b = id / groups;

    int goff = group_offset[g] * spatial;
    int boff = b * stride;

    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}