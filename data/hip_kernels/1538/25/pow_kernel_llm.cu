#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Compute global index for this thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we are within bounds
    if(i < N) 
    {
        // Efficient memory access by caching ALPHA in a register
        float alpha = ALPHA;
        
        // Compute power and store result
        Y[i*INCY] = powf(X[i*INCX], alpha);
    }
}