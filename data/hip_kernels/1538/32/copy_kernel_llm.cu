#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Cache the grid and block dimensions for better readability
    int gridSize = blockDim.x * gridDim.x;
    
    // Loop through all elements that the current thread can process
    for (int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x; i < N; i += gridSize) {
        Y[i*INCY + OFFY] = X[i*INCX + OFFX];
    }
}