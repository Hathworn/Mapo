#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D grid for simplicity
    if(idx < n){
        float diff = truth[idx] - pred[idx];
        error[idx] = diff * diff;
        delta[idx] = diff;
    }
}