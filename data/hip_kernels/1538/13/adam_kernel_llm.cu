#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        // Precompute powers outside of the loop for efficiency
        float B1_pow_t = powf(B1, t);
        float B2_pow_t = powf(B2, t);
        
        // Use one division and precompute common terms
        float inv_one_minus_B1_pow_t = 1.f / (1.f - B1_pow_t);
        float inv_one_minus_B2_pow_t = 1.f / (1.f - B2_pow_t);

        // Use local variables for better performance
        float mhat = m[index] * inv_one_minus_B1_pow_t;
        float vhat = v[index] * inv_one_minus_B2_pow_t;
        
        // Update x using efficient computation
        x[index] += rate * mhat / (sqrtf(vhat) + eps);
    }
}