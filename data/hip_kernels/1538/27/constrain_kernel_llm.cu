#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved constraint kernel for better performance
__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate thread stride

    for (int i = idx; i < N; i += stride) { // Loop to cover entire data set
        X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i * INCX])); // Apply constraints
    }
}