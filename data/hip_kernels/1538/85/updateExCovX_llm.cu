#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateExCovX(double *e_x_cov_x, double gauss_d2, int valid_voxel_num)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Reducing number of operations with addition in loop condition
    while (id < valid_voxel_num) {
        e_x_cov_x[id] *= gauss_d2;
        id += stride;
    }
}