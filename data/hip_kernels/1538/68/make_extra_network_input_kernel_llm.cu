#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void make_extra_network_input_kernel(float* dev_x_coors_for_sub, float* dev_y_coors_for_sub, float* dev_num_points_per_pillar, float* dev_x_coors_for_sub_shaped, float* dev_y_coors_for_sub_shaped, float* dev_pillar_feature_mask, const int MAX_NUM_POINTS_PER_PILLAR) {
    int ith_pillar = blockIdx.x;
    int ith_point = threadIdx.x;
    int ind = ith_pillar * MAX_NUM_POINTS_PER_PILLAR + ith_point;
    
    // Cache x and y as shared for potential use across threads
    float x = dev_x_coors_for_sub[ith_pillar];
    float y = dev_y_coors_for_sub[ith_pillar];
    
    // Simplify and compact memory writes with combined assignments
    dev_x_coors_for_sub_shaped[ind] = x;
    dev_y_coors_for_sub_shaped[ind] = y;
    
    int num_points_for_a_pillar = dev_num_points_per_pillar[ith_pillar];
    
    // Use conditional operator for a concise mask assignment
    dev_pillar_feature_mask[ind] = (ith_point < num_points_for_a_pillar) ? 1.0f : 0.0f;
}