#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using threadIdx.y to process multiple elements per thread
__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Compute stride for grid-stride loop

    for (int i = idx; i < n; i += stride) { // Use grid-stride loop to handle more elements
        c[i] += a[i] * b[i];
    }
}