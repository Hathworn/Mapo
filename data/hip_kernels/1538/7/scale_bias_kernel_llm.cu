#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int filter = blockIdx.y;
    int batch = blockIdx.z;
    int tid = threadIdx.x;
    int block_size = blockDim.x;
    int offset = blockIdx.x * block_size;

    for (int idx = offset + tid; idx < size; idx += block_size) { // Optimize with loop for unrolled offsets
        output[(batch * n + filter) * size + idx] *= biases[filter];
    }
}