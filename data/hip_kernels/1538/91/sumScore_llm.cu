#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumScore(double *score, int full_size, int half_size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < half_size; i += stride) {
        // Use conditional operator within the index check to prevent unnecessary memory accesses
        int neighbor_index = i + half_size;
        if (neighbor_index < full_size) {
            score[i] += score[neighbor_index];
        }
    }
}