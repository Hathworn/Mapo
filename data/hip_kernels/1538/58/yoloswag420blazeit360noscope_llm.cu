#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplify 1D indexing computation
    int stride = gridDim.x * blockDim.x; // Calculate stride for grid-stride loop
    for (int i = id; i < size; i += stride) { // Use grid-stride loop for better coalescing and efficiency
        input[i] = (rand[i] < prob) ? 0 : input[i] * scale; // Conditional assignment
    }
}