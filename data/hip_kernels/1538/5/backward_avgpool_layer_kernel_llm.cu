#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    // Calculate the global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Returned early if out of bounds
    if(id >= n) return;

    // Calculate spatial location
    int k = id % c;
    int b = id / c;

    int out_index = k + c * b;
    float scale_factor = 1.0f / (w * h); // Precompute scale factor
    int base_in_index = h * w * (k + b * c); // Precompute base in_index

    for(int i = 0; i < w * h; ++i) {
        int in_index = base_in_index + i;
        in_delta[in_index] += out_delta[out_index] * scale_factor; // Use multiplication instead of division
    }
}