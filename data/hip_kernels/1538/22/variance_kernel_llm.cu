#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize variance kernel
__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Precompute constants outside loop for efficiency
    float scale = 1.f/(batch * spatial - 1);
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

    // Avoid redundant computations by ensuring i is within bounds
    if (i >= filters) return;

    // Start with zero variance
    float var = 0.0f;

    // Compute variance using pre-computed mean
    for(int j = 0; j < batch; ++j){
        for(int k = 0; k < spatial; ++k){
            int index = j*filters*spatial + i*spatial + k;
            float diff = x[index] - mean[i];
            var += diff * diff;  // Use inline computation instead of powf
        }
    }
    
    // Save the computed value after scaling
    variance[i] = var * scale;
}