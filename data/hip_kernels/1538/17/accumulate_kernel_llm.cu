#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    float temp_sum = 0.0f; // Use a local variable for accumulation
    for (int k = 0; k < n; ++k){
        temp_sum += x[k * groups + i];
    }
    sum[i] = temp_sum; // Write the result back to global memory
}