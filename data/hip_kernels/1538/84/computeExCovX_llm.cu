#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeExCovX(float *trans_x, float *trans_y, float *trans_z, int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *centr_x, double *centr_y, double *centr_z, double gauss_d1, double gauss_d2, double *e_x_cov_x, double *icov00, double *icov01, double *icov02, double *icov10, double *icov11, double *icov12, double *icov20, double *icov21, double *icov22)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Loop unrolling and pre-computation optimization
    for (int i = id; i < valid_points_num; i += stride) {
        int pid = valid_points[i];
        double d_x = static_cast<double>(trans_x[pid]);
        double d_y = static_cast<double>(trans_y[pid]);
        double d_z = static_cast<double>(trans_z[pid]);

        for (int j = starting_voxel_id[i]; j < starting_voxel_id[i + 1]; j++) {
            int vid = voxel_id[j];

            double t_x = d_x - centr_x[vid];
            double t_y = d_y - centr_y[vid];
            double t_z = d_z - centr_z[vid];

            // Pre-compute repeated calculations for efficiency
            double icov0 = t_x * icov00[vid] + t_y * icov01[vid] + t_z * icov02[vid];
            double icov1 = t_x * icov10[vid] + t_y * icov11[vid] + t_z * icov12[vid];
            double icov2 = t_x * icov20[vid] + t_y * icov21[vid] + t_z * icov22[vid];

            e_x_cov_x[j] = exp(-gauss_d2 * (icov0 * t_x + icov1 * t_y + icov2 * t_z) / 2.0);
        }
    }
}