#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Precompute values to reduce redundant calculations
    int w_h = w * h;
    int w_h_c = w_h * c;

    // Calculate indices with fewer operations
    int in_index = i;
    int b = i / w_h_c;
    int in_c = (i / w_h) % c;
    int in_h = (i / w) % h;
    int in_w = i % w;

    int out_c = c / (stride * stride);
    int offset = in_c / out_c;
    int c2 = in_c % out_c;

    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;

    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    // Minimized conditional checks
    if (forward)
        out[out_index] = x[in_index];
    else
        x[in_index] = out[out_index];
}