#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation

    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;  // Early exit for redundant calculations

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;

    // Loop unrolling to improve performance
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int base = b * size * n + i;  // Precompute base index
            int i1 = base + f1 * size;
            int i2 = base + f2 * size;
            float output_i1 = output[i1];
            float output_i2 = output[i2];
            sum += output_i1 * output_i2;
            norm1 += output_i1 * output_i1;
            norm2 += output_i2 * output_i2;
        }
    }
    
    norm1 = rsqrtf(norm1); // Use reciprocal square root for faster computation
    norm2 = rsqrtf(norm2); // Use reciprocal square root for faster computation
    float norm_inv = norm1 * norm2; // Compute inverse of norm directly

    sum *= norm_inv;
    float scale_sum = -scale * sum;

    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int base = b * size * n + i;  // Precompute base index
            int i1 = base + f1 * size;
            int i2 = base + f2 * size;
            float output_i1 = output[i1];
            float output_i2 = output[i2];
            delta[i1] += scale_sum * output_i2 * norm_inv;
            delta[i2] += scale_sum * output_i1 * norm_inv;
        }
    }
}