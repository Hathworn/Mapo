#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global thread index directly, combining grid and block computation
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x; // Calculate total threads
    // Use stride loop to cover entire array if total threads < N
    for (; i < N; i += total_threads) {
        Y[i * INCY] *= X[i * INCX];
    }
}