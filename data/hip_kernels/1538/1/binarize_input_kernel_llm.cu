#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    // Compute the global index for the thread
    int s = blockIdx.x * blockDim.x + threadIdx.x;
    if (s >= size) return; // Ensure within bounds

    float mean = 0.0f;

    // Use shared memory for mean accumulation
    __shared__ float shared_mean[1024];
    float local_sum = 0.0f;

    // Step through each sample for this thread and accumulate mean
    for (int i = 0; i < n; ++i) {
        local_sum += fabsf(input[i * size + s]);
    }

    // Store local sum in shared memory and synchronize
    shared_mean[threadIdx.x] = local_sum;
    __syncthreads();

    // Use a single thread to compute the mean from shared memory
    if (threadIdx.x == 0) {
        float block_sum = 0.0f;
        for (int j = 0; j < blockDim.x; ++j) {
            block_sum += shared_mean[j];
        }
        mean = block_sum / (n * blockDim.x);
    }

    // Broadcast the computed mean value to all threads
    __syncthreads();
    if (threadIdx.x == 0) {
        shared_mean[0] = mean;
    }
    __syncthreads();
    mean = shared_mean[0];

    // Binarize inputs based on the mean value
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}