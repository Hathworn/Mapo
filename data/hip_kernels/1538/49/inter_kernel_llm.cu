#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation
    int totalThreads = gridDim.x * blockDim.x; // Calculate total number of threads

    for (int i = idx; i < (NX + NY) * B; i += totalThreads) { // Use loop to allow other threads to work concurrently
        int b = i / (NX + NY);
        int j = i % (NX + NY);
        if (j < NX) {
            OUT[i] = X[b * NX + j];
        } else {
            OUT[i] = Y[b * NY + j - NX];
        }
    }
}