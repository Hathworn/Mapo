#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation for 1D grid
    int stride = gridDim.x * blockDim.x;            // Calculate the stride for loop unrolling

    for (int idx = i; idx < n; idx += stride) {     // Loop to cover all elements in steps of stride
        if (mask[idx] == mask_num) x[idx] = val;    // Only update if the mask condition is met
    }
}