#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id < size) { // Efficient check to ensure thread is within bounds
        int i = id % minw;
        int j = (id / minw) % minh;
        int k = (id / (minw * minh)) % minc;
        int b = id / (minw * minh * minc);

        int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
        int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));
        out[out_index] = s1 * out[out_index] + s2 * add[add_index]; // Perform operation more succinctly
    }
}