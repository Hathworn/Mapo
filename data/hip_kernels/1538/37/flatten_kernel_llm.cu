#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate the global index of the element
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread processes valid data
    if (index >= N) return;
    
    // Compute the indices using modulus and division
    int in_s = index % spatial;
    index = index / spatial;
    int in_c = index % layers;
    index = index / layers;
    int b = index;

    // Compute flat index positions for source and destination
    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;

    // Forward or backward path
    if (forward) 
        out[i2] = x[i1];
    else 
        out[i1] = x[i2];
}