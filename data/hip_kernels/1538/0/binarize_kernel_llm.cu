#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Compute global thread index using single blockDim.y multiplication for optimization
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        binary[i] = (x[i] >= 0) ? 1.0f : -1.0f; // Use float literals for clarity
    }
}