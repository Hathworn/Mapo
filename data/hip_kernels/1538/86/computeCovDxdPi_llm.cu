#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeCovDxdPi(int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *inverse_covariance, int voxel_num, double gauss_d1, double gauss_d2, double *point_gradients, double *cov_dxd_pi, int valid_voxel_num)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int row = blockIdx.y;
    int col = blockIdx.z;

    // Process only if row and col within bounds
    if (row < 3 && col < 6) {
        // Directly compute starting offsets for faster access
        double *icov0 = inverse_covariance + row * 3 * voxel_num;
        double *cov_dxd_pi_tmp = cov_dxd_pi + (row * 6 + col) * valid_voxel_num;
        double *pg_tmp0 = point_gradients + col * valid_points_num;
        double *pg_tmp1 = pg_tmp0 + 6 * valid_points_num;
        double *pg_tmp2 = pg_tmp1 + 6 * valid_points_num;

        for (int i = id; i < valid_points_num; i += stride) {
            double pg0 = pg_tmp0[i];
            double pg1 = pg_tmp1[i];
            double pg2 = pg_tmp2[i];

            for (int j = starting_voxel_id[i]; j < starting_voxel_id[i + 1]; j++) {
                int vid = voxel_id[j];

                // Preload inverse_covariance values to reduce L1 cache misses
                double ic0 = icov0[vid];
                double ic1 = icov0[vid + voxel_num];
                double ic2 = icov0[vid + 2 * voxel_num];

                cov_dxd_pi_tmp[j] = ic0 * pg0 + ic1 * pg1 + ic2 * pg2;
            }
        }
    }
}