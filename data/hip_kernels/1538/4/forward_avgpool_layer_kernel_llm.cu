#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int spatial_area = w * h;
    int k = id % c;
    int b = id / c;

    int out_index = k + c * b;
    float sum = 0.0f;
    
    // Unroll loop for potential performance gain
    for(int i = 0; i < spatial_area; ++i) {
        int in_index = i + spatial_area * (k + b * c);
        sum += input[in_index];
    }
    output[out_index] = sum / spatial_area; // Calculate average
}