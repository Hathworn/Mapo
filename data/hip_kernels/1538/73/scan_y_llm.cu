#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan_y(int *g_odata, int *g_idata, int n)
{
    extern __shared__ int temp[];
    int thid = threadIdx.x;
    int bid = blockIdx.x;
    int bdim = blockDim.x;
    int gdim = gridDim.x;
    int offset = 1;

    // Coalesced global memory access
    int ai_global = bid + (2*thid) * gdim;
    int bi_global = ai_global + gdim;

    temp[2*thid]   = g_idata[ai_global];
    temp[2*thid+1] = g_idata[bi_global];

    for (int d = n>>1; d > 0; d >>= 1) {
        __syncthreads();
        if (thid < d) {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (thid == 0) { temp[n - 1] = 0; }

    for (int d = 1; d < n; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (thid < d) {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();

    // Coalesced writing back to global memory
    g_odata[ai_global] = temp[2*thid+1];
    g_odata[bi_global] = (2*thid+2 == bdim*2) ? temp[2*thid+1] + g_idata[bi_global] : temp[2*thid+2];
}