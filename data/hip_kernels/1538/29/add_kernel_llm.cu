#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify indexing to 1D grid
    int stride = blockDim.x * gridDim.x;            // Calculate stride

    for (; i < N; i += stride) {                     // Use loop for memory coalescing
        X[i * INCX] += ALPHA;                        // Vectorized addition
    }
}