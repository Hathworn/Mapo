#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Use increment to simplify index calculation and get rid of gridDim.y multiplication
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll loop for improved memory access pattern
    for (; i < n; i += blockDim.x * gridDim.x) {
        float t = truth[i];
        float p = pred[i];
        error[i] = (t) ? -logf(p) : 0.0f; // Use logf for single-precision float operations
        delta[i] = t - p;
    }
}