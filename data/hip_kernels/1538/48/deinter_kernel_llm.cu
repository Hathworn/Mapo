#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < (NX + NY) * B){
        int b = i / (NX + NY);
        int j = i % (NX + NY);

        // Use shared memory to enhance memory access patterns
        __shared__ float sharedOUT[1024];
        sharedOUT[threadIdx.x] = OUT[i];
        __syncthreads();

        if (j < NX){
            if(X) atomicAdd(&(X[b * NX + j]), sharedOUT[threadIdx.x]); // Use atomic to ensure correct sum
        } else {
            if(Y) atomicAdd(&(Y[b * NY + j - NX]), sharedOUT[threadIdx.x]); // Use atomic to ensure correct sum
        }
    }
}