#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Compute global index
    int gridStride = blockDim.x * gridDim.x;        // Calculate grid stride
    for (; i < n; i += gridStride) {                // Loop with grid stride
        if (mask[i] == mask_num) {                  // Check condition
            x[i] *= scale;                          // Scale if condition is true
        }
    }
}