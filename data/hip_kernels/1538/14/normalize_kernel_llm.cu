#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    // Calculate the global index using a single dimension for thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a loop to prevent unused threads
    for (; index < N; index += blockDim.x * gridDim.x) {
        int f = (index / spatial) % filters;

        // Optimize computation: precompute sqrtf value
        float inv_stddev = rsqrtf(variance[f] + 1e-5f);

        // Apply normalization using the precomputed value
        x[index] = (x[index] - mean[f]) * inv_stddev;
    }
}