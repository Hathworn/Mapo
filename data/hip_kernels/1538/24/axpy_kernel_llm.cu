#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Compute unique thread index.
    int stride = gridDim.x * blockDim.x;           // Calculate stride for grid-stride loop.
    for (; i < N; i += stride) {                    // Use grid-stride loop for better utilization.
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}