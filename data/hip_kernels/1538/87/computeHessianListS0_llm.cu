#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeHessianListS0(float *trans_x, float *trans_y, float *trans_z, int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *centroid_x, double *centroid_y, double *centroid_z, double *icov00, double *icov01, double *icov02, double *icov10, double *icov11, double *icov12, double *icov20, double *icov21, double *icov22, double *point_gradients, double *tmp_hessian, int valid_voxel_num)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int col = blockIdx.y;

    if (col < 6) {
        double *tmp_pg0 = point_gradients + col * valid_points_num;
        double *tmp_pg1 = tmp_pg0 + 6 * valid_points_num;
        double *tmp_pg2 = tmp_pg1 + 6 * valid_points_num;
        double *tmp_h = tmp_hessian + col * valid_voxel_num;

        for (int i = id; i < valid_points_num; i += stride) {
            // Precompute invariant elements for efficiency
            int pid = valid_points[i];
            double d_x = static_cast<double>(trans_x[pid]);
            double d_y = static_cast<double>(trans_y[pid]);
            double d_z = static_cast<double>(trans_z[pid]);

            double pg0 = tmp_pg0[i];
            double pg1 = tmp_pg1[i];
            double pg2 = tmp_pg2[i];

            for (int j = starting_voxel_id[i]; j < starting_voxel_id[i + 1]; j++) {
                int vid = voxel_id[j];

                // Cache the icov matrix for the current voxel
                double ic00 = icov00[vid], ic01 = icov01[vid], ic02 = icov02[vid];
                double ic10 = icov10[vid], ic11 = icov11[vid], ic12 = icov12[vid];
                double ic20 = icov20[vid], ic21 = icov21[vid], ic22 = icov22[vid];

                // Compute components of the equation to reduce redundant operations
                double t_x = (d_x - centroid_x[vid]);
                double t_y = (d_y - centroid_y[vid]);
                double t_z = (d_z - centroid_z[vid]);

                // Accumulate result into the hessian matrix
                tmp_h[j] = (t_x * (ic00 * pg0 + ic01 * pg1 + ic02 * pg2))
                        + (t_y * (ic10 * pg0 + ic11 * pg1 + ic12 * pg2))
                        + (t_z * (ic20 * pg0 + ic21 * pg1 + ic22 * pg2));
            }
        }
    }
}