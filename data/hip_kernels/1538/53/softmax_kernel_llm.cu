#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;

    // Use parallel reduction to find the largest element
    for(i = 0; i < n; ++i){
        float val = input[i*stride];
        largest = (val > largest) ? val : largest;
    }

    for(i = 0; i < n; ++i){
        float e = expf(input[i*stride]/temp - largest/temp);
        sum += e;
        output[i*stride] = e;
    }

    // Normalize outputs in a separate loop to avoid bank conflicts
    for(i = 0; i < n; ++i){
        output[i*stride] /= sum;
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    // Calculate global thread ID more efficiently
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (id >= batch * groups) return;

    // Use integer division and modulus with ID directly
    int b = id / groups;
    int g = id % groups;
    
    softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}