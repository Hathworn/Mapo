#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate 1D index for the thread
    int stride = blockDim.x * gridDim.x; // Calculate stride for grid-stride loop

    for (int i = idx; i < n; i += stride) { // Use grid-stride loop for better workload distribution
        float diff = truth[i] - pred[i];
        error[i] = abs(diff);
        delta[i] = (diff > 0) ? 1.0f : -1.0f; // Ensure floating-point arithmetic
    }
}