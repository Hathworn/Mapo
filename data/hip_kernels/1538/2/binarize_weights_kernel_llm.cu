#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
    if (f >= n) return;

    float mean = 0.0f;
    int offset = f * size;

    // Use shared memory to accumulate the sum for enhanced performance
    extern __shared__ float shared_sum[];
    shared_sum[threadIdx.x] = 0.0f;

    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_sum[threadIdx.x] += fabsf(weights[offset + i]);
    }
    __syncthreads();

    // Reduce the sum in shared memory
    int block_size = blockDim.x;
    while (block_size > 1) {
        int half = (block_size + 1) >> 1; 
        if (threadIdx.x < half && (threadIdx.x + half) < blockDim.x) {
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + half];
        }
        __syncthreads();
        block_size = half;
    }

    if (threadIdx.x == 0) {
        mean = shared_sum[0] / size;
    }
    __syncthreads();

    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[offset + i] = (weights[offset + i] > 0) ? mean : -mean;
    }
}