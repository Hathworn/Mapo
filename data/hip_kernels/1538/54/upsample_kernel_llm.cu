#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    // Calculate the global thread index
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    // Early return if the index goes beyond bounds
    if(i >= N) return;

    // Calculate output index mapping
    int out_index = i;
    int out_w = out_index % (w * stride);
    int out_h = (out_index / (w * stride)) % (h * stride);
    int out_c = (out_index / (w * stride * h * stride)) % c;
    int b = (out_index / (w * stride * h * stride * c)) % batch;

    // Calculate input index mapping
    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_index = (b * c + out_c) * (w * h) + in_h * w + in_w;

    // Perform the forward or backward pass operation
    if(forward)
        out[out_index] += scale * x[in_index];
    else
        atomicAdd(x + in_index, scale * out[out_index]);
}