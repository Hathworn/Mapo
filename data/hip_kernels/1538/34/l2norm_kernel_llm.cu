#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    // Use a single flattened index for unique identification
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    int b = index / spatial;  // Batch index
    int i = index % spatial;  // Spatial index

    float sum = 0.0f;
    
    // Calculate the sum of squares
    for (int f = 0; f < filters; ++f) {
        int idx = b * filters * spatial + f * spatial + i;
        float val = x[idx];
        sum += val * val;  // Avoid powf for better performance
    }

    sum = sqrtf(sum);
    if (sum == 0.0f) sum = 1.0f;

    // Normalize x and compute dx
    for (int f = 0; f < filters; ++f) {
        int idx = b * filters * spatial + f * spatial + i;
        x[idx] /= sum;
        dx[idx] = (1.0f - x[idx]) / sum;
    }
}