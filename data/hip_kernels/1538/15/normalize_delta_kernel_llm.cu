#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if (index >= N) return;

    int stride = gridDim.x * blockDim.x;
    for (; index < N; index += stride) {
        int f = (index / spatial) % filters;

        // Calculate the normalization coefficients
        float inv_sqrt_var = 1.f / sqrtf(variance[f] + 0.00001f);
        float coeff1 = variance_delta[f] * 2.f / (spatial * batch);
        float coeff2 = mean_delta[f] / (spatial * batch);

        // Apply transformation
        delta[index] = delta[index] * inv_sqrt_var + coeff1 * (x[index] - mean[f]) + coeff2;
    }
}