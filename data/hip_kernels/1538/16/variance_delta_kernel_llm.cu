#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta) 
{
    // Calculate linear index for filters
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Initialize variance_delta
    float var_delta_sum = 0.0f;
    
    // Pre-compute common values for better performance
    float mean_i = mean[i];
    float variance_val = variance[i] + 0.00001f;
    float factor = -.5f * powf(variance_val, (float)(-3.f/2.f));

    // Loop through batch and spatial dimensions
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            var_delta_sum += delta[index] * (x[index] - mean_i);
        }
    }
    
    // Apply factor to the sum
    variance_delta[i] = var_delta_sum * factor;
}