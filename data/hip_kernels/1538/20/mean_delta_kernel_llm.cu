#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;  // Ensure thread is within bounds
    int j, k;
    float sum = 0; // Use local variable to reduce global memory access
    for (j = 0; j < batch; ++j) {
        for (k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += delta[index]; // Accumulate in local variable
        }
    }
    mean_delta[i] = sum * (-1.f / sqrtf(variance[i] + .00001f)); // Compute and store the result
}