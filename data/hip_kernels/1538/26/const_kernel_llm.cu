#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Compute the unique thread index.
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Parallelize the assignment over threads in steps of total threads.
    for (; i < N; i += gridDim.x * blockDim.x) {
        X[i * INCX] = ALPHA;
    }
}