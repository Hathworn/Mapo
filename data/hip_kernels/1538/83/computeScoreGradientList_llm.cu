#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeScoreGradientList(float *trans_x, float *trans_y, float *trans_z, int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *centroid_x, double *centroid_y, double *centroid_z, int voxel_num, double *e_x_cov_x, double *cov_dxd_pi, double gauss_d1, int valid_voxel_num, double *score_gradients)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int col = blockIdx.y;

    if (col < 6) {
        double *sg = score_gradients + col * valid_points_num;
        double *cov_dxd_pi_mat0 = cov_dxd_pi + col * valid_voxel_num;
        double *cov_dxd_pi_mat1 = cov_dxd_pi_mat0 + 6 * valid_voxel_num;
        double *cov_dxd_pi_mat2 = cov_dxd_pi_mat1 + 6 * valid_voxel_num;

        // Iterate only over valid points within the given stride
        for (int i = id; i < valid_points_num; i += stride) {
            int pid = valid_points[i];
            double d_x = static_cast<double>(trans_x[pid]);
            double d_y = static_cast<double>(trans_y[pid]);
            double d_z = static_cast<double>(trans_z[pid]);

            double tmp_sg = 0.0;

            // Unroll computation to prevent repetitive memory access
            for (int j = starting_voxel_id[i]; j < starting_voxel_id[i + 1]; j++) {
                int vid = voxel_id[j];
                double tmp_ex = e_x_cov_x[j];

                // Avoid invalid computations by checking for NaN and range
                if (!(tmp_ex > 1.0 || tmp_ex < 0.0 || tmp_ex != tmp_ex)) {
                    tmp_ex *= gauss_d1;

                    double centroid_dx = d_x - centroid_x[vid];
                    double centroid_dy = d_y - centroid_y[vid];
                    double centroid_dz = d_z - centroid_z[vid];

                    tmp_sg += (centroid_dx * cov_dxd_pi_mat0[j] + 
                               centroid_dy * cov_dxd_pi_mat1[j] + 
                               centroid_dz * cov_dxd_pi_mat2[j]) * tmp_ex;
                }
            }

            // Store result in score_gradients
            sg[i] = tmp_sg;
        }
    }
}