#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    // Calculate unique thread index for this kernel
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform operation only if within bounds
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}
```
