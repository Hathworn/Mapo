#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void testKernel(float *g_idata, float *g_odata)
{
    // shared memory
    // the size is determined by the host application
    extern __shared__ float sdata[];

    // access thread id and number of threads in this block
    const unsigned int tid = threadIdx.x;
    const unsigned int num_threads = blockDim.x;
    const unsigned int gid = blockIdx.x * blockDim.x + tid;

    // read in input data from global memory
    sdata[tid] = g_idata[gid];
    __syncthreads();

    // perform some computations
    sdata[tid] *= (float)num_threads;
    __syncthreads();

    // write data to global memory
    g_odata[gid] = sdata[tid];
}