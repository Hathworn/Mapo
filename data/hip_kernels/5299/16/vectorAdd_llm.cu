#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Unroll loop iterations for better performance
    #pragma unroll
    for (int idx = i; idx < numElements; idx += blockDim.x * gridDim.x) {
        C[idx] = A[idx] + B[idx];
    }
}