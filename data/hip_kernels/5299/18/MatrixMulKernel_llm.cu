#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width) {
    // Determine starting points for the tiles
    int Mstart = Width * BLOCK_SIZE * blockIdx.y;
    int Nstart = BLOCK_SIZE * blockIdx.x;
    float temp = 0;

    // Loop through tiles
    for (int m = Mstart, n = Nstart; m <= Mstart + Width - 1; m += BLOCK_SIZE, n += BLOCK_SIZE * Width) {
        // Load data into shared memory using coalesced access
        __shared__ float Ms[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Ns[BLOCK_SIZE][BLOCK_SIZE];

        Ms[threadIdx.y][threadIdx.x] = M[m + Width * threadIdx.y + threadIdx.x];
        Ns[threadIdx.y][threadIdx.x] = N[n + Width * threadIdx.y + threadIdx.x];
        
        __syncthreads();

        // Use unrolled loop for better ILP (Instruction-Level Parallelism)
        #pragma unroll
        for (int i = 0; i < BLOCK_SIZE; ++i) {
            temp += Ms[threadIdx.y][i] * Ns[i][threadIdx.x];
        }
        
        __syncthreads();
    }

    // Store the result back to the matrix P
    P[Width * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x + Width * threadIdx.y + threadIdx.x] = temp;
}