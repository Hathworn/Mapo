#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define tileSize 32

//function for data initialization
void initialization( double *M,  double *N, int arow, int acol, int brow, int bcol);
//(for Debugging) prints out the input data
void printInput( double *M,  double *N, int arow, int acol, int brow,  int bcol);
//(for Debugging) prints out the output data
void printOutput( double *P_C,  double *P_G, int arow, int bcol);

//GPU kernels

__global__ void vectorAddition(const double *A, const double *B, double *C, int numElements)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = gridDim.x * blockDim.x;

    // Ensure index is within bounds
    while (index < numElements)
    {
        C[index] = A[index] + B[index];
        index += totalThreads;  // Move index by total threads
    }
}