#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16  // Define a tile width for shared memory usage

__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width) {
    // Shared memory declaration for optimization
    __shared__ float M_shared[TILE_WIDTH][TILE_WIDTH];
    __shared__ float N_shared[TILE_WIDTH][TILE_WIDTH];

    int Row = blockIdx.y * blockDim.y + threadIdx.y; // Calculate the row index of the P element and M
    int Col = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the column index of P and N

    float Pvalue = 0.0;
    
    // Loop over tiles
    for (int m = 0; m < (Width + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
        // Load elements into shared memory for M and N
        if (Row < Width && m * TILE_WIDTH + threadIdx.x < Width)
            M_shared[threadIdx.y][threadIdx.x] = M[Row * Width + m * TILE_WIDTH + threadIdx.x];
        else
            M_shared[threadIdx.y][threadIdx.x] = 0.0;

        if (Col < Width && m * TILE_WIDTH + threadIdx.y < Width)
            N_shared[threadIdx.y][threadIdx.x] = N[(m * TILE_WIDTH + threadIdx.y) * Width + Col];
        else
            N_shared[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads(); // Synchronize threads to ensure shared memory loads are done

        // Compute each thread's partial dot product
        for (int k = 0; k < TILE_WIDTH; ++k) {
            Pvalue += M_shared[threadIdx.y][k] * N_shared[k][threadIdx.x];
        }

        __syncthreads(); // Synchronize before loading new data into shared memory
    }

    // Write back the result to global memory
    if (Row < Width && Col < Width) {
        P[Row * Width + Col] = Pvalue;
    }
}