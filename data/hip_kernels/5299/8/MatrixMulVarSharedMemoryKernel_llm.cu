#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE tileSize

__global__ void MatrixMulVarSharedMemoryKernel(float* M, float* N, float* P, int widthAHeightB, int heightA, int widthB) {
    int Mstart = widthAHeightB * TILE_SIZE * blockIdx.y;
    int Mend = Mstart + widthAHeightB - 1;
    int mstep = TILE_SIZE;
    int Nstart = TILE_SIZE * blockIdx.x;
    int nstep = TILE_SIZE * widthB;
    float temp = 0;

    __shared__ float Ms[TILE_SIZE][TILE_SIZE];
    __shared__ float Ns[TILE_SIZE][TILE_SIZE];

    // optimized: remove branching by using for loop conditions directly
    for (int m = Mstart, n = Nstart; m <= Mend; m += mstep, n += nstep) {
        int mPos = m + widthAHeightB * threadIdx.y + threadIdx.x;
        int nPos = n + widthB * threadIdx.y + threadIdx.x;

        Ms[threadIdx.y][threadIdx.x] = (m <= Mend && mPos < widthAHeightB * heightA) ? M[mPos] : 0.0f;
        Ns[threadIdx.y][threadIdx.x] = (n <= Mend && nPos < widthB * widthAHeightB) ? N[nPos] : 0.0f;
        
        __syncthreads();

        for (int i = 0; i < TILE_SIZE; ++i) {
            temp += Ms[threadIdx.y][i] * Ns[i][threadIdx.x];
        }
        __syncthreads();
    }

    // optimized: single if-statement for P calculation
    int pPos = widthB * TILE_SIZE * blockIdx.y + TILE_SIZE * blockIdx.x + widthB * threadIdx.y + threadIdx.x;
    if (blockIdx.y * TILE_SIZE + threadIdx.y < heightA && blockIdx.x * TILE_SIZE + threadIdx.x < widthB) {
        P[pPos] = temp;
    }
}