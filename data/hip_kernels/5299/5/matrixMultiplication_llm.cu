#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define tileSize 32

//function for data initialization
void initialization(double* M, double* N, int arow, int acol, int brow, int bcol);
//(for Debugging) prints out the input data
void printInput(double* M, double* N, int arow, int acol, int brow, int bcol);
//(for Debugging) prints out the output data
void printOutput(double* P_C, double* P_G, int arow, int bcol);

__global__ void matrixMultiplication(double* M, double* N, double* P, int widthAHeightB, int heightA, int widthB) {
    int row = blockIdx.y * tileSize + threadIdx.y;
    int col = blockIdx.x * tileSize + threadIdx.x;

    double temp = 0.0;
    __shared__ double Ms[tileSize][tileSize];
    __shared__ double Ns[tileSize][tileSize];

    for (int t = 0; t < (widthAHeightB + tileSize - 1) / tileSize; ++t) {
        if (row < heightA && (t * tileSize + threadIdx.x) < widthAHeightB)
            Ms[threadIdx.y][threadIdx.x] = M[row * widthAHeightB + t * tileSize + threadIdx.x];
        else
            Ms[threadIdx.y][threadIdx.x] = 0.0;

        if (col < widthB && (t * tileSize + threadIdx.y) < widthAHeightB)
            Ns[threadIdx.y][threadIdx.x] = N[(t * tileSize + threadIdx.y) * widthB + col];
        else
            Ns[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        for (int i = 0; i < tileSize; ++i) {
            temp += Ms[threadIdx.y][i] * Ns[i][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < heightA && col < widthB) {
        P[row * widthB + col] = temp;
    }
}