#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define tileSize 32

//function for data initialization
void initialization( double *M,  double *N, int arow, int acol, int brow, int bcol);
//(for Debugging) prints out the input data
void printInput( double *M,  double *N, int arow, int acol, int brow,  int bcol);
//(for Debugging) prints out the output data
void printOutput( double *P_C,  double *P_G, int arow, int bcol);

//GPU kernels

__global__ void matrixTransposeSqr(double *P, double* M, int width, int height)
{
    unsigned int xIdx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int yIdx = blockDim.y * blockIdx.y + threadIdx.y;

    // Use shared memory tiles for better memory coalescing
    __shared__ double tile[tileSize][tileSize];

    // Check for bounds and load data into shared memory
    if (xIdx < width && yIdx < height) {
        unsigned int inIdx = xIdx + width * yIdx;
        tile[threadIdx.y][threadIdx.x] = M[inIdx];
    }
    __syncthreads();

    // Transpose using shared memory
    xIdx = blockDim.y * blockIdx.y + threadIdx.x;
    yIdx = blockDim.x * blockIdx.x + threadIdx.y;

    if (xIdx < height && yIdx < width) {
        unsigned int outIdx = yIdx + height * xIdx;
        P[outIdx] = tile[threadIdx.x][threadIdx.y];
    }
}