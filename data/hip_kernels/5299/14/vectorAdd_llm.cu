#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    // Pre-calculate offset for the current thread
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Use condition to directly calculate if within bounds
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}