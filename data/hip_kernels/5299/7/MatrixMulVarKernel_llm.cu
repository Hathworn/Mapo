#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulVarKernel(float* M, float* N, float* P, int widthAHeightB, int heightA, int widthB) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y; // Calculate row index
    int Col = blockIdx.x * blockDim.x + threadIdx.x; // Calculate column index
    if ((Row < heightA) && (Col < widthB)) {
        float Pvalue = 0.0f;
        int indexM = Row * widthAHeightB; // Pre-compute row start index for M
        int indexN = Col; // Column start index for N
        for (int k = 0; k < widthAHeightB; ++k) {
            Pvalue += M[indexM + k] * N[k * widthB + indexN]; // Accumulate product
        }
        P[Row * widthB + Col] = Pvalue; // Store result
    }
}