#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const double *A, const double *B, double *C, int numElements)
{
    // Cache thread index and global index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Perform addition if within bounds, removing unnecessary debugging
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}