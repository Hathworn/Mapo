#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Assertion to check for errors
__global__ void kernel_trialDiv(long* n, int* r) {
    int bx = blockIdx.x;  // ID thread
    int tx = threadIdx.x;

    // Identify the row and column of the Pd element to work on
    long memIndex = bx * TILE_WIDTH + tx;
    long n_value = n[memIndex];  // Load data to a register

    // Unrolling the loop for better performance
    #pragma unroll
    for (int i = 0; i < 256; i++) {
        if (n_value % d_sPrimes[i] == 0) {
            r[memIndex]--;  // Efficient decrement
        }
    }
}