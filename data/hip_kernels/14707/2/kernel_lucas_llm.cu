#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_lucas(long* nArray, long* dArray, int* rArray, long len) {
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    
    // Compute global thread index
    long memIndex = bx * TILE_WIDTH + tx;
    if (memIndex >= len) return; // Boundary check, avoids unnecessary thread computations

    long long d = (long long) dArray[memIndex];
    long long n = (long long) nArray[memIndex];
    long long q = (1 - d) / 4;
    long long u = 0, v = 2, u2 = 1, v2 = 1, q2 = 2 * q;
    long long t = (n + 1) / 2;
    int length = 32 - __clz(t);

    // Use a single loop to minimize the branch costs and potential instruction stalls
    for (int i = 0; i < length; ++i) {
        u2 = (u2 * v2) % n;
        v2 = (v2 * v2 - q2) % n;

        if (t & 1) {
            long long uTemp = u;
            u = (u2 * v + u * v2);
            v = v2 * v + u2 * uTemp * d;
            if (u & 1) u += n;
            u = (u / 2) % n;
            if (v & 1) v += n;
            v = (v / 2) % n;
        }

        q = (q * q) % n;
        q2 = q + q;
        t >>= 1;
    }

    rArray[memIndex] = (u == 0);
}