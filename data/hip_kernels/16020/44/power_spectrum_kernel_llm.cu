#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    // Calculate global and shared index
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int block_offset = blockIdx.x * ldi;

    // Set up pointers for input and output arrays
    float *Ar = A_in + block_offset;
    float *Aw = A_out + blockIdx.x * ldo;

    if (global_idx == 0 || global_idx >= row_length / 2) return; // Skip special case and out of bound threads

    // Optimized calculation for power spectrum excluding special cases
    float2 val = reinterpret_cast<float2 *>(Ar)[global_idx];
    float ret = val.x * val.x + val.y * val.y;
    Aw[global_idx] = use_power ? ret : sqrtf(ret);

    // Single thread block handles special case
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];
        Aw[0] = use_power ? real * real : fabsf(real);
        Aw[row_length / 2] = use_power ? im * im : fabsf(im);
    }
}