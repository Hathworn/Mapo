#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_lifter_and_floor_energy( int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (int frame = blockIdx.x; frame < num_frames; frame += gridDim.x) {
        float *feats = features + frame * ldf;

        // Apply lifter coefficients using shared memory
        if (cepstral_lifter != 0.0f) {
            for (int c = tid; c < num_cols; c += blockDim.x * gridDim.x) {
                float lift = lifter_coeffs[c];
                float f = feats[c];
                feats[c] = f * lift;
            }
        }

        // Use one thread per block to handle energy
        if (use_energy && threadIdx.x == 0) {
            float energy = log_energy[frame];
            float log_energy_floor = log(energy_floor);

            if (energy_floor > 0.0f && energy < log_energy_floor) {
                energy = log_energy_floor;
            }
            feats[0] = energy;
        }
    }
}