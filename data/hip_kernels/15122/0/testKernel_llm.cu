#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifndef _matrix_KERNEL_H_
#define _matrix_KERNEL_H_

#define SDATA(index) CUT_BANK_CHECKER(sdata, index)

__global__ void testKernel(float* d_matrixA, float* d_matrixB, float* d_matrixC, const unsigned int ah, const unsigned int aw, const unsigned int bh, const unsigned int bw) {
    // Shared memory optimization
    #ifdef CHANGE4
    __shared__ float shm_matrixB[KERNEL_SIZE + (2 * KERNEL_LENGTH)];
    #elif defined(CHANGE1)
    __shared__ float shm_matrixB[KERNEL_SIZE];
    #endif

    #ifdef CHANGE4
    __shared__ float shm_subMatrixA[BLOCK_SIZE_HEIGHT * BLOCK_SIZE_WIDTH + WARP_SIZE];
    #elif defined(CHANGE3)
    __shared__ float shm_subMatrixA0[BLOCK_SIZE_HEIGHT * BLOCK_SIZE_WIDTH];
    __shared__ float shm_subMatrixA1[BLOCK_SIZE_HEIGHT * BLOCK_SIZE_WIDTH];
    #elif defined(CHANGE2)
    __shared__ float shm_subMatrixA[2 * BLOCK_SIZE_HEIGHT * BLOCK_SIZE_WIDTH];
    #endif

    const unsigned int bx = blockIdx.x;
    const unsigned int by = blockIdx.y;

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    #ifdef CHANGE3
    int xstep = bx;
    int ystep = 2 * by;
    #elif defined(CHANGE2)
    int xstep = bx;
    int ystep = by;
    #else
    int xstep = BLOCK_SIZE * bx;
    int ystep = BLOCK_SIZE * by;
    #endif

    #ifdef CHANGE3
    float sum0 = 0;
    float sum1 = 0;
    #else
    float sum = 0;
    #endif

    int y = ystep + ty;
    int x = xstep + tx;

    #ifdef CHANGE4
    // Initialize shared memory and pad with zeros
    if (tx < KERNEL_LENGTH) {
        shm_matrixB[tx] = 0;
        shm_matrixB[KERNEL_SIZE + tx] = 0;
    }

    if (tx < KERNEL_SIZE)
        shm_matrixB[tx + KERNEL_LENGTH] = d_matrixB[tx];

    if (tx < WARP_SIZE)
        shm_subMatrixA[tx] = 0;

    __syncthreads();

    #elif defined(CHANGE1)
    if (tx < KERNEL_SIZE)
        shm_matrixB[tx] = d_matrixB[tx];
    #endif

    /* -------------------------------- Computation -------------------------------------*/

    #ifdef CHANGE4
    for (int j = 0; j < bh + 1; j++) {
        shm_subMatrixA[tx + WARP_SIZE] = 0;
        if ((y - j + 1) > -1) {
            shm_subMatrixA[tx + WARP_SIZE] = d_matrixA[(y - j + 1) * aw + x];
        }
        __syncthreads();

        for (int k = 0; k < bw; ++k) {
            float b0 = shm_matrixB[j * bw + k];
            float b1 = shm_matrixB[(j + 1) * bw + k];
            float a = shm_subMatrixA[tx - k + WARP_SIZE];
            sum0 += a * b0;
            sum1 += a * b1;
        }
        __syncthreads();
    }

    #elif defined(CHANGE3)
    for (int j = 0; j < bh; j++) {
        if ((y - j) > -1 && (y - j) < ah) {
            shm_subMatrixA0[tx] = d_matrixA[(y - j) * aw + x];
        }
        if ((y + 1 - j) > -1 && (y + 1 - j) < ah) {
            shm_subMatrixA1[tx] = d_matrixA[(y + 1 - j) * aw + x];
        }
        __syncthreads();

        for (int k = 0; k < bw; ++k) {
            float b = shm_matrixB[j * bw + k];
            float a0 = shm_subMatrixA0[tx - k];
            float a1 = shm_subMatrixA1[tx - k];
            if ((y - j) > -1 && (y - j) < ah && (x - k) > -1 && (x - k) < aw) {
                sum0 += a0 * b;
            }
            if ((y + 1 - j) > -1 && (y + 1 - j) < ah && (x - k) > -1 && (x - k) < aw) {
                sum1 += a1 * b;
            }
        }
        __syncthreads();
    }

    #elif defined(CHANGE2)
    for (int j = 0; j < bh; j++) {
        if ((y - j) > -1 && (y - j) < ah) {
            shm_subMatrixA[tx] = d_matrixA[(y - j) * aw + x];
        }
        __syncthreads();

        for (int k = 0; k < bw; ++k) {
            float b = shm_matrixB[j * bw + k];
            float a = shm_subMatrixA[tx - k];
            if ((y - j) > -1 && (y - j) < ah && (x - k) > -1 && (x - k) < aw) {
                sum += a * b;
            }
        }
        __syncthreads();
    }

    #elif defined(CHANGE1)
    for (int j = 0; j < bh; j++) {
        for (int k = 0; k < bw; ++k) {
            float b = shm_matrixB[j * bw + k];
            if ((y - j) > -1 && (y - j) < ah && (x - k) > -1 && (x - k) < aw) {
                float a = d_matrixA[(y - j) * aw + (x - k)];
                sum += a * b;
            }
        }
    }
    __syncthreads();

    #else
    for (int j = 0; j < bh; j++) {
        for (int k = 0; k < bw; ++k) {
            float b = d_matrixB[j * bw + k];
            if ((y - j) > -1 && (y - j) < ah && (x - k) > -1 && (x - k) < aw) {
                float a = d_matrixA[(y - j) * aw + (x - k)];
                sum += a * b;
            }
        }
    }
    #endif

    #ifdef CHANGE4
    d_matrixC[(1 * y * aw) + x] = sum0;
    d_matrixC[(((1 * y) + 1) * aw) + x] = sum1;
    #elif defined(CHANGE3)
    d_matrixC[(1 * y * aw) + x] = sum0;
    d_matrixC[(((1 * y) + 1) * aw) + x] = sum1;
    #else
    d_matrixC[y * aw + x] = sum;
    #endif
}
#endif // #ifndef _matrix_KERNEL_H_