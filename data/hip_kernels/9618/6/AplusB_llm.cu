#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AplusB(int *ret, int a, int b) {
    // Optimize by computing sum once and reusing the result
    int result = a + b;
    ret[threadIdx.x] = result + threadIdx.x;
}