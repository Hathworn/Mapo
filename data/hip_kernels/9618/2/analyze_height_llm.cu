#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
cudaStructTest
testing/optimizing how to access/manipulate/return
structures in cuda.
*/

#define N 30
#define TRUE 1
#define FALSE 0
#define MAX_BLOCKS 65000

typedef struct{
    int id;
    int age;
    int height;
} Person;

// Optimize analyze_height kernel
__global__ void analyze_height(Person *people, int *statResults)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread is within bounds
    if (id < N) {
        // Access person directly without a temporary variable
        statResults[id] = (people[id].height != 6) ? 1 : 0;
    }
}