#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Constants and Macro Definitions
#define N 30
#define TRUE 1
#define FALSE 0
#define MAX_BLOCKS 65000

// Struct Definition
typedef struct {
    int id;
    int age;
    int height;
} Person;

// Optimized global kernel function
__global__ void analyze_id(Person *people, int *statResults) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure only valid threads perform operations
    if (id < N) {
        statResults[id] = (people[id].id < 6) ? 1 : 0; // Directly assign result with condition
    }
}