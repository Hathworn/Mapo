#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 30
#define TRUE 1
#define FALSE 0
#define MAX_BLOCKS 65000

int cuda_setup(int computeCapability);

typedef struct {
    int id;
    int age;
    int height;
} Person;

__global__ void analyze_age(Person *people, int *statResults) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Check if thread id is within bounds before accessing memory
    if (id < N) {
        Person person = people[id];
        
        // Use ternary operator for simplicity
        statResults[id] = (person.age != 28) ? 1 : 0;
    }
}