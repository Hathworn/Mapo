#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelGetPhi3(const int N, double *T, double *q)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        double qi = q[i];  // Load q[i] into register for reuse
        T[i] = qi * qi * qi;  // Compute cube using register value
    }
}