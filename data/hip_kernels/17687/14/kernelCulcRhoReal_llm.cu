#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelCulcRhoReal(const int N, double *rho, double *q, double *p, const double lambda, const double g)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        double qi = q[i];
        double pi = p[i];
        double qi2 = qi * qi; // Precompute qi squared

        rho[i] = 0.5 * qi2;
        rho[i] += 0.5 * pi * pi;
        double qi4 = qi2 * qi2; // Precompute qi to the fourth power
        rho[i] += (lambda / 4.0) * qi4;
        rho[i] += (g / 6.0) * qi4 * qi2; // Use qi4 and qi2 for sixth power
    }
}