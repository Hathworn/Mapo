#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelGetOmega(const int N, double *omega, double *kSqr, 
                               const double sigma2, const double sigma4, 
                               const double lambda, const double g)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check within bounds and pre-calculate constants for efficiency
    if (i < N)
    {
        const double constant = 3 * lambda * sigma2 + 15 * g * sigma4;
        omega[i] = sqrt(1.0 + kSqr[i] + constant);
    }
}