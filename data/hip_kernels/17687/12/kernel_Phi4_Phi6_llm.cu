#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_Phi4_Phi6(const int N, double *t, double *q, const double lambda, const double g)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds and minimize floating-point operations
    if (i < N)
    {
        double qi = q[i];
        double qi_square = qi * qi;
        t[i] = qi_square * qi * (lambda + g * qi_square);
    }
}