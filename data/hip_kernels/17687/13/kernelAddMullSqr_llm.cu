#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize the kernel function using block-level parallel reduction technique
__global__ void kernelAddMullSqr(const int N, double* S, double* A, double m)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to minimize global memory access
    __shared__ double sharedMem[256];  // Adjust size based on blockDim.x

    // Perform the computation if within bounds
    if (i < N)
    {
        sharedMem[threadIdx.x] = m * A[i] * A[i];
    }
    else
    {
        sharedMem[threadIdx.x] = 0.0;  // Handle out-of-bounds threads
    }

    // Synchronize threads to ensure shared memory is fully populated
    __syncthreads();

    // Use a loop for block-level parallel reduction
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            sharedMem[threadIdx.x] += sharedMem[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Update the global memory result
    if (threadIdx.x == 0)
    {
        atomicAdd(&S[blockIdx.x], sharedMem[0]);
    }
}