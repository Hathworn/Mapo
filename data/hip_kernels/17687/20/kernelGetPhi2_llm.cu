#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelGetPhi2(const int N, double *T, double *q)
{
    // Use shared memory for faster access
    extern __shared__ double shared_q[];
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int ti = threadIdx.x;

    // Load data into shared memory
    if (i < N) {
        shared_q[ti] = q[i];
        __syncthreads();
        
        // Perform the computation using shared memory
        T[i] = shared_q[ti] * shared_q[ti];
    }
}