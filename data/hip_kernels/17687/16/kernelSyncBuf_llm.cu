#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelSyncBuf(double *A, double *A0)
{
    const int i = threadIdx.x;
    const int j = threadIdx.y;
    const int k = threadIdx.z;
    const int N1 = blockDim.x;
    const int N2 = blockDim.y;
    const int N3 = blockDim.z;

    const int iB = blockIdx.x;
    const int jB = blockIdx.y;
    const int kB = blockIdx.z;
    const int N2B = gridDim.y;
    const int N3B = gridDim.z;

    const int iG = i + iB * N1;
    const int jG = j + jB * N2;
    const int kG = k + kB * N3;
    const int N2G = N2 * N2B;
    const int N3G = N3 * N3B;

    const int indB = k + N3 * (j + N2 * i);
    const int indA = kB + N3B * (jB + N2B * iB);
    const int indA0 = kG + N3G * (jG + N2G * iG);

    extern __shared__ double B[];
    B[indB] = A0[indA0];
    __syncthreads();

    // Use shared memory for reduction
    for (int offset = 1; offset < N1 * N2 * N3; offset *= 2) {
        if (indB % (2 * offset) == 0) {
            B[indB] += B[indB + offset];
        }
        __syncthreads();
    }

    if (indB == 0) {
        A[indA] = B[0] / (N1 * N2 * N3);
    }
}