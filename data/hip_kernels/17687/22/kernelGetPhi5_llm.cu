#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelGetPhi5(const int N, double *T, double *q)
{
    // Use a shared memory to optimize accessing q while calculating phi5
    extern __shared__ double shared_q[];
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load q into shared memory for faster access
    if (i < N)
    {
        shared_q[threadIdx.x] = q[i];
        __syncthreads();

        // Compute phi5 with reduced global memory access
        double q_val = shared_q[threadIdx.x]; // Access from shared memory
        T[i] = q_val * q_val * q_val * q_val * q_val;
    }
}