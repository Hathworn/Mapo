#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void createRaysOrthoKernel(float4* rays, int width, int height, float x0, float y0, float z, float dx, float dy, unsigned rayMask )
{
    int rayx = blockIdx.x * blockDim.x + threadIdx.x;
    int rayy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the current thread is within bounds
    if (rayx < width && rayy < height) {
        float tMinOrMask = rayMask ? __int_as_float(rayMask) : 0.0f;

        int idx = rayx + rayy * width;
        
        // Calculate ray origin and direction efficiently
        rays[2 * idx] = make_float4(x0 + rayx * dx, y0 + rayy * dy, z, tMinOrMask);
        rays[2 * idx + 1] = make_float4(0.0f, 0.0f, 1.0f, 1e34f);
    }
}