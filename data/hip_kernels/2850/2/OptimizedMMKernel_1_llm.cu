#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 32

__global__ void OptimizedMMKernel_1(float *a, float *b, float *c, int size)
{
    __shared__ float sharedA[TILE_SIZE][TILE_SIZE];
    __shared__ float sharedB[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = blockIdx.x * blockDim.x + tx;
    int y = blockIdx.y * blockDim.y + ty;

    float sum = 0;
    int tilesPerGrid = size / TILE_SIZE;

    for (int i = 0; i < tilesPerGrid; i++)
    {
        // Load elements into shared memory; leverage memory coalescing
        sharedA[ty][tx] = a[y * size + i * TILE_SIZE + tx];
        sharedB[ty][tx] = b[(i * TILE_SIZE + ty) * size + x];

        __syncthreads();  // Synchronize after loading tiles

        #pragma unroll  // Unroll inner loop for better performance
        for (int j = 0; j < TILE_SIZE; j++)
        {
            sum += sharedA[ty][j] * sharedB[j][tx];
        }

        __syncthreads();  // Synchronize to ensure completion of all operations
    }
    
    // Store computed value into the output matrix
    c[y * size + x] = sum;
}