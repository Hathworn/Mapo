#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 32

__global__ void OptimizedMMKernel_0(float *a, float *b, float *c, int size)
{
    // Shared memory to store the tiles of matrices A and B
    __shared__ float sharedA[TILE_SIZE][TILE_SIZE];
    __shared__ float sharedB[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int x = blockIdx.x * blockDim.x + tx;
    int y = blockIdx.y * blockDim.y + ty;

    float sum = 0.0f;

    int tilesPerGrid = size / TILE_SIZE;
    for (int i = 0; i < tilesPerGrid; i++)
    {
        // Each thread loads one element of the A and B tiles.
        sharedA[ty][tx] = a[y * size + (i * TILE_SIZE + tx)];
        sharedB[ty][tx] = b[(i * TILE_SIZE + ty) * size + x];

        // Synchronize to make sure the tile is loaded before using it
        __syncthreads();

        #pragma unroll // Unroll for loop for performance
        for (int j = 0; j < TILE_SIZE; j++)
        {
            sum += sharedA[ty][j] * sharedB[j][tx];
        }

        // Synchronize before computing next tile
        __syncthreads();
    }

    // Write the result to the output matrix C
    c[y * size + x] = sum;
}