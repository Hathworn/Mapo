#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 32

__global__ void OptimizedMMKernel_2_32(float *a, float *b, float *c, int size)
{
    __shared__ float sharedA[TILE_SIZE][TILE_SIZE];
    __shared__ float sharedB[TILE_SIZE][TILE_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int x = blockIdx.x * blockDim.x + tx;
    int y = blockIdx.y * blockDim.y + ty;

    float sum = 0;

    int tilesPerGrid = size / TILE_SIZE;
    for (int i = 0; i < tilesPerGrid; i++)
    {
        // Load elements from global to shared memory
        sharedA[ty][tx] = a[y * size + (i * TILE_SIZE + tx)];
        sharedB[ty][tx] = b[(i * TILE_SIZE * size) + ty * size + x];

        __syncthreads();  // Ensure all threads have loaded the matrices

        #pragma unroll  // Unroll loop for performance improvement
        for (int j = 0; j < TILE_SIZE; j++) {
            sum += sharedA[ty][j] * sharedB[j][tx];
        }
        
        __syncthreads();  // Ensure computation using current tile is done
    }

    c[y * size + x] = sum;  // Write result to global memory
}