#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 32

__global__ void OptimizedMMKernel_2_16(float *a, float *b, float *c, int size)
{
    __shared__ float sharedA[16][16];
    __shared__ float sharedB[16][16];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int x = blockIdx.x * blockDim.x + tx;
    int y = blockIdx.y * blockDim.y + ty;

    float sum = 0.0f;

    // Use loop unrolling to load data into shared memory and compute the sum
    int tilesPerGrid = size / blockDim.x;
    for (int i = 0; i < tilesPerGrid; ++i)
    {
        sharedA[ty][tx] = a[y * size + i * 16 + tx];
        sharedB[ty][tx] = b[(i * 16 * size) + (ty * size) + x];

        __syncthreads();

        #pragma unroll // Unrolling the loop for performance improvement
        for (int j = 0; j < 16; ++j) {
            sum += sharedA[ty][j] * sharedB[j][tx];
        }

        __syncthreads();
    }

    c[y * size + x] = sum;
}