#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 32

__global__ void OptimizedMMKernel_2_4(float *a, float *b, float *c, int size)
{
    // Create shared matrices for rows of A and columns of B
    __shared__ float sharedA[4][4];
    __shared__ float sharedB[4][4];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int x = blockIdx.x * blockDim.x + tx;
    int y = blockIdx.y * blockDim.y + ty;

    float sum = 0;

    // Optimize tile calculation to reduce division
    int tilesPerGrid = size >> 2; 

    for (int i = 0; i < tilesPerGrid; i++)
    {
        // Each thread loads element into A and B
        sharedA[ty][tx] = a[(y * size) + (i << 2) + tx];  // Use bitshift for multiplication
        sharedB[ty][tx] = b[(i << 2) * size + (ty * size) + x]; // Use bitshift for multiplication

        // Wait for all threads to load section of the shared matrices
        __syncthreads();
        
        // Unroll sum updates
        sum += sharedA[ty][0] * sharedB[0][tx];  
        sum += sharedA[ty][1] * sharedB[1][tx];
        sum += sharedA[ty][2] * sharedB[2][tx];
        sum += sharedA[ty][3] * sharedB[3][tx];

        // Wait for all threads to compute their partial sum before next tile
        __syncthreads();
    }

    // Store the full sum as the result
    c[y * size + x] = sum;
}