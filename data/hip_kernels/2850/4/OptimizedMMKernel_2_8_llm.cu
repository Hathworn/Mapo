#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/// Tile size used by the OptimizedMMKernel
#define TILE_SIZE 32

/// Optimized matrix multiplication CUDA Kernel
__global__ void OptimizedMMKernel_2_8(float *a, float *b, float *c, int size)
{
    // Create shared matrices for rows of A and columns of B
    __shared__ float sharedA[8][8];
    __shared__ float sharedB[8][8];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int x = blockIdx.x * blockDim.x + tx;
    int y = blockIdx.y * blockDim.y + ty;

    float sum = 0;

    // Improved loop unrolling for accessing shared memory
    int tilesPerGrid = size / blockDim.x;
    for (int i = 0; i < tilesPerGrid; i++)
    {
        // Each thread loads element into A and B
        sharedA[ty][tx] = a[(y * size) + (i * 8) + tx];
        sharedB[ty][tx] = b[(i * 8 * size) + (ty * size) + x];

        // Wait for all threads to load each section of the shared matrix
        __syncthreads();

        #pragma unroll
        for (int k = 0; k < 8; ++k)
        {
            sum += sharedA[ty][k] * sharedB[k][tx];
        }

        // Wait for all threads to compute their partial sum before loading the next
        __syncthreads();
    }

    // Store the full sum as the result
    c[y * size + x] = sum;
}