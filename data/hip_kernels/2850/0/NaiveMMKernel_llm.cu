#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/// Tile size used by the OptimizedMMKernel
#define TILE_SIZE 32

/// Prints a matrix out to the stderr stream
__global__ void NaiveMMKernel(float *a, float *b, float *c, int size)
{
    int xOut = blockDim.x * blockIdx.x + threadIdx.x;
    int yOut = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if thread is within matrix dimensions
    if (xOut < size && yOut < size) {
        float outValue = 0;
        
        // Using shared memory for a and b
        __shared__ float aTile[TILE_SIZE][TILE_SIZE];
        __shared__ float bTile[TILE_SIZE][TILE_SIZE];

        for (int i = 0; i < size; i += TILE_SIZE) {
            // Load data into shared memory
            aTile[threadIdx.y][threadIdx.x] = a[yOut * size + (i + threadIdx.x)];
            bTile[threadIdx.y][threadIdx.x] = b[(i + threadIdx.y) * size + xOut];
            
            __syncthreads();  // Synchronize to ensure all data is loaded

            // Perform partial dot product
            for (int j = 0; j < TILE_SIZE; ++j) {
                outValue += aTile[threadIdx.y][j] * bTile[j][threadIdx.x];
            }

            __syncthreads();  // Synchronize after computation
        }

        // Store sum of dot products in C matrix
        c[yOut * size + xOut] = outValue;
    }
}