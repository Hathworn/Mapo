#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatSub(float *A, float *B, float *C, int nx, int ny) {
    // Calculate the global thread ID
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = iy * nx + ix; // Optimize for coalesced memory access

    // Ensure indices are within bounds
    if (ix < nx && iy < ny) {
        C[idx] = A[idx] - B[idx];
    }
}