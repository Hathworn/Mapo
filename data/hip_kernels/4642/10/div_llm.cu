#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void div(float *A, float *B, float *C, int nx, int ny) {
    // Calculate the global thread index for a 2D grid
    int idx_x = threadIdx.x + blockIdx.x * blockDim.x;
    int idx_y = threadIdx.y + blockIdx.y * blockDim.y;

    // Linear indexing considering 2D grid.
    int idx = idx_x * ny + idx_y;

    // Perform division only if within bounds
    if (idx_x < nx && idx_y < ny) {
        C[idx] = A[idx] / B[idx];
    }
}