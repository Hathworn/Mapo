#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatAdd(float *A, float *B, float *C, int nx, int ny) {
    // Calculate the linear index using blockIdx, blockDim, and threadIdx
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = ix * ny + iy;

    // Ensure thread index is within matrix bounds before performing addition
    if (ix < nx && iy < ny) {
        C[idx] = A[idx] + B[idx];
    }
}