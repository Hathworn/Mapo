#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void div(float *A, float *B, float *C, int nx, int ny) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = ix * ny + iy;

    // Ensure the thread is within the bounds
    if (ix < nx && iy < ny) {
        C[idx] = A[idx] / B[idx];
    }
}