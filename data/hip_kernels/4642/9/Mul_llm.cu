#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Mul(float *A, float *B, float *C, int nx, int ny){
    // Compute ix and iy once and reuse
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    // Optimize condition check for valid index
    if (ix < nx && iy < ny) {
        int idx = ix * ny + iy;
        // Perform multiplication only for valid elements
        C[idx] = A[idx] * B[idx];
    }
}
```
