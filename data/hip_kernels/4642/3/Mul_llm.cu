#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Mul(float *A, float *B, float *C, int nx, int ny) {
    // Calculate flattened index once per thread
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = ix * ny + iy;

    // Use a conditional branch to avoid unnecessary calculations
    if (ix < nx && iy < ny) {
        C[idx] = A[idx] * B[idx];
    }
}