#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMul(float *a, float *b, float *c, int m, int n, int k)
{
    // Calculate global row and column indexes
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure row and column are within matrix bounds
    if (row < m && col < k)
    {
        float sum = 0.0f; // Use float for precision
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col]; // Perform operation with float
        }
        c[row * k + col] = sum; // Assign computed sum to the result matrix
    }
}