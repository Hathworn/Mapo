#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixTranspose(float *a, float *b, int nx, int ny) {
    // Calculate the column and row indices using shared memory
    __shared__ float tile[32][32]; // Assuming block size is 32 for better memory coalescing
    
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (ix < nx && iy < ny) {
        // Use local thread indices for improved memory access pattern
        int idx = ix * ny + iy;
        tile[threadIdx.y][threadIdx.x] = a[idx];
    }
    
    __syncthreads(); // Synchronize threads before writing transposed element

    // Transpose elements using shared memory and store them
    ix = blockIdx.y * blockDim.y + threadIdx.x;
    iy = blockIdx.x * blockDim.x + threadIdx.y;

    if (ix < ny && iy < nx) {
        int odx = iy * ny + ix;
        b[odx] = tile[threadIdx.x][threadIdx.y];
    }
}