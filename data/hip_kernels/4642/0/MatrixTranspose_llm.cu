#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved version of MatrixTranspose kernel function
__global__ void MatrixTranspose(float *a, float *b, int nx, int ny) {
    // Utilize built-in variables to calculate global index more efficiently
    int ix = blockIdx.x * blockDim.x + threadIdx.x; 
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Early exit if index is out of bounds
    if (ix >= nx || iy >= ny) return;

    // Calculate linear indices for input and output matrices
    int idx = ix * ny + iy;
    int odx = iy * nx + ix;

    // Perform the transpose
    b[odx] = a[idx];
}