#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMul(float *a, float *b, float *c, int m, int n, int k)
{
    // Use shared memory for block-level matrix multiplication
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    for (int t = 0; t < (n - 1) / TILE_SIZE + 1; ++t) {
        // Load data into shared memory
        if (row < m && t*TILE_SIZE + threadIdx.x < n)
            tileA[threadIdx.y][threadIdx.x] = a[row * n + t*TILE_SIZE + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < k && t*TILE_SIZE + threadIdx.y < n)
            tileB[threadIdx.y][threadIdx.x] = b[(t*TILE_SIZE + threadIdx.y) * k + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        // Perform block-level matrix multiplication
        for (int i = 0; i < TILE_SIZE; ++i) {
            sum += tileA[threadIdx.y][i] * tileB[i][threadIdx.x];
        }

        __syncthreads();
    }

    // Store the result
    if (row < m && col < k) {
        c[row * k + col] = sum;
    }
}