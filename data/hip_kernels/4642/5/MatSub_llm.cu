#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatSub(float *A, float *B, float *C, int nx, int ny) {
    // Calculate unique index for each element
    int idx = blockIdx.x * blockDim.x * ny + threadIdx.x * ny + blockIdx.y * blockDim.y + threadIdx.y;

    // Check boundary before subtraction
    if (idx < nx * ny) {
        C[idx] = A[idx] - B[idx];
    }
}