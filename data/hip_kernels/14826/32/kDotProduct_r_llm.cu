#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numCols, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];
    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;
    if (eidx < numCols) {
        for (; eidx < numElements; eidx += numCols) {
            shmem[threadIdx.x] += a[eidx] * b[eidx];
        }
    }
    __syncthreads();

    // Perform reduction using warp shuffle to optimize sync and memory access
    for (int offset = DP_BLOCKSIZE / 2; offset > 0; offset /= 2) {
        shmem[threadIdx.x] += __shfl_down_sync(0xFFFFFFFF, shmem[threadIdx.x], offset);
    }
    
    // Write result to target, ensuring only the first thread writes
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}