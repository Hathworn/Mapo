#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate flattened index for current thread
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    
    // Use shared memory to reduce global memory access (when applicable)
    __shared__ float sharedSrc[BLOCK_SIZE]; // Adjust BLOCK_SIZE as necessary
    
    // Calculate total number of elements
    const uint totalElements = tgtWidth * tgtHeight;
    
    // Loop over elements handled by this thread
    for (uint i = idx; i < totalElements; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        
        // Load relevant source data into shared memory
        if (threadIdx.x < srcWidth) {
            sharedSrc[threadIdx.x] = src[srcY * srcWidth + threadIdx.x];
        }
        __syncthreads(); // Synchronize to ensure data is loaded
        
        // Perform the tiling operation using shared memory
        tgt[i] = sharedSrc[srcX];
    }
}