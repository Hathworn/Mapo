#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    // Launch fewer threads if out of bounds to improve performance
    if (tidx < maxNumThreads) {
        /* Each thread gets same seed, a different sequence number,
        no offset */
        hiprand_init(seed, tidx, 0, &state[tidx]);
    }
}