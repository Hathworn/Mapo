#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Copy_double(double *a, double *b, size_t len)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Unroll loop for better parallelism and memory access pattern
    for(; idx < len; idx += blockDim.x * gridDim.x) {
        b[idx] = a[idx];
    }
}