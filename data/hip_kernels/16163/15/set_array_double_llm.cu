#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_array_double(double *a, double value, size_t len)
{
    // Use strided loop for memory coalescing and better performance
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    for (size_t i = idx; i < len; i += stride) {
        a[i] = value;
    }
}