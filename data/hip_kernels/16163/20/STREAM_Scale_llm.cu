#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Scale(float *a, float *b, float scale, size_t len)
{
    // Calculate the unique global thread index
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Utilize grid-stride loop for processing elements
    for (size_t i = idx; i < len; i += blockDim.x * gridDim.x) {
        b[i] = scale * a[i];
    }
}