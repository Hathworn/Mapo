#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function for vector addition
__global__ void STREAM_Add_Optimized(float *a, float *b, float *c, size_t len) {
    // Calculate global index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Process only valid indices
    if (idx < len) {
        c[idx] = a[idx] + b[idx];
    }
}