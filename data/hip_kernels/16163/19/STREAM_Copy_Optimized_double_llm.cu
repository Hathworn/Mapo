#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Copy_Optimized_double(double *a, double *b, size_t len)
{
    // Calculate global thread index once
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use grid-stride loop for better occupancy and performance
    for (; idx < len; idx += blockDim.x * gridDim.x)
    {
        b[idx] = a[idx];
    }
}