#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Triad_Optimized_double(double *a, double *b, double *c, double scalar, size_t len)
{
    // Calculate global thread ID
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Use stride loop to handle larger data sets efficiently
    size_t stride = blockDim.x * gridDim.x;
    
    for (; idx < len; idx += stride) {
        c[idx] = a[idx] + scalar * b[idx];
    }
}