#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Triad_Optimized(float *a, float *b, float *c, float scalar, size_t len)
{
    // Use all available threads by removing the size check
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len) 
    {
        // Perform computation for each valid index
        c[idx] = a[idx] + scalar * b[idx];
    }
}