#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Add(float *a, float *b, float *c, size_t len)
{
    // Calculate global thread index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Perform the addition only if index is within bounds
    if (idx < len) {
        c[idx] = a[idx] + b[idx];
    }
}