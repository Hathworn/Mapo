#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Copy_Optimized(float *a, float *b, size_t len)
{
    // Calculate the global index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Check and assign elements within bounds
    if (idx < len) {
        b[idx] = a[idx];
    }
}