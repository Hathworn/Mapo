#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Scale_Optimized_double(double *a, double *b, double scale, size_t len)
{
    // Calculate global thread index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use grid-stride loop to utilize more threads
    for (; idx < len; idx += blockDim.x * gridDim.x) {
        b[idx] = scale * a[idx];
    }
}