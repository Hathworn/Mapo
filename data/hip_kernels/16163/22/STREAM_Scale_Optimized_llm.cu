#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Scale_Optimized(float *a, float *b, float scale, size_t len)
{
    // Calculate the unique thread index within the grid
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Use stride to enable processing of remaining elements
    size_t stride = blockDim.x * gridDim.x;

    // Loop to process beyond the initial assignment if len is larger
    for (size_t i = idx; i < len; i += stride)
    {
        b[i] = scale * a[i];
    }
}