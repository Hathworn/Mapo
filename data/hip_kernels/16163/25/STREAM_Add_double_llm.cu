#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Add_double(double *a, double *b, double *c, size_t len)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Use a for loop for better readability and maintenance
    for (; idx < len; idx += blockDim.x * gridDim.x) {
        c[idx] = a[idx] + b[idx];
    }
}