#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Triad(float *a, float *b, float *c, float scalar, size_t len)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;

    // Unroll the loop for optimization
    for (; idx + 3 < len; idx += 4 * stride) {
        c[idx] = a[idx] + scalar * b[idx];
        c[idx + stride] = a[idx + stride] + scalar * b[idx + stride];
        c[idx + 2 * stride] = a[idx + 2 * stride] + scalar * b[idx + 2 * stride];
        c[idx + 3 * stride] = a[idx + 3 * stride] + scalar * b[idx + 3 * stride];
    }

    // Handle remaining elements
    for (; idx < len; idx += stride) {
        c[idx] = a[idx] + scalar * b[idx];
    }
}