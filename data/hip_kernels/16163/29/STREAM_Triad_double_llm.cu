#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Triad_double(double *a, double *b, double *c, double scalar, size_t len)
{
    // Calculate the global thread ID
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride to handle larger arrays and ensure all threads are utilized
    size_t stride = blockDim.x * gridDim.x;

    // Loop through elements that each thread will process
    for (; idx < len; idx += stride) {
        c[idx] = a[idx] + scalar * b[idx];
    }
}