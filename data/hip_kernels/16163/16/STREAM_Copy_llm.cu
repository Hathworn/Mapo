#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Copy(float *a, float *b, size_t len)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x; // Pre-calculate stride for efficiency
    for (; idx < len; idx += stride) {      // Use for loop for better readability and optimization
        b[idx] = a[idx];
    }
}