#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_prng(hiprandState *states, const unsigned long int seed)
{
    // Use combined indexing to improve scalability beyond a single block
    const int t_idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, t_idx, 0, &states[t_idx]);
}