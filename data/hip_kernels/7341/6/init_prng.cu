#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_prng(hiprandState *states, const unsigned long int seed)
{
const int t_idx = threadIdx.x;
hiprand_init(seed, t_idx, 0, &states[t_idx]);
}