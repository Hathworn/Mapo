#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mandelKernel(double planoFactorXd, double planoFactorYd, double planoVxd, double planoVyd, int maxIteracionesd, unsigned int *coloresd, int img_width, int img_height, int num_processes, int my_pid, int rw) {

    // Determine pixel
    int columna = blockIdx.x * blockDim.x + threadIdx.x;
    int fila = (rw * MAX_ROWS_PER_KERNEL) + blockIdx.y * blockDim.y + threadIdx.y;
    int real_row = fila * num_processes + my_pid;

    if(real_row >= img_height)
        return;

    // Pre-compute often used values
    double fila_factor = planoFactorYd * ((double)(img_height - 1) - (double)real_row) + planoVyd;
    double columna_factor = planoFactorXd * (double)columna + planoVxd;

    // Real pixel coords
    double X = columna_factor;
    double Y = fila_factor;

    double pReal = 0.0, pImag = 0.0, pRealAnt, pImagAnt;
    double distancia;
    int i = 0;

    // Use a single loop instead of do-while
    for (i = 0; i < maxIteracionesd; ++i) {
        pRealAnt = pReal;
        pImagAnt = pImag;
        pReal = (pRealAnt * pRealAnt) - (pImagAnt * pImagAnt) + X;
        pImag = 2.0 * pRealAnt * pImagAnt + Y;
        distancia = pReal * pReal + pImag * pImag;
        if (distancia > 4.0) // Break early if condition is met
            break;
    }

    // Assign color value directly
    coloresd[fila * img_width + columna] = (i == maxIteracionesd) ? 0 : i; 
}