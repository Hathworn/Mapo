#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mandelKernel(double planoFactorXd, double planoFactorYd, double planoVxd, double planoVyd, int maxIteracionesd, unsigned int *coloresd, int img_width, int img_height, int num_processes, int my_pid) {
    int columna = blockIdx.x * blockDim.x + threadIdx.x;
    int fila = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check bounds to prevent unnecessary computations
    if (columna >= img_width || fila >= img_height) return;

    int real_row = (fila * num_processes) + my_pid;
    double X = (planoFactorXd * (double)columna) + planoVxd;
    double Y = (planoFactorYd * ((double)(img_height - 1) - (double)real_row)) + planoVyd;
    double pReal = 0.0;
    double pImag = 0.0;
    double distancia;
    int i = 0;

    // Simplified computation with fewer operations
    while (i < maxIteracionesd && (distancia = pReal * pReal + pImag * pImag) <= 4.0) {
        double pRealAnt = pReal;
        double pImagAnt = pImag;
        pReal = ((pRealAnt * pRealAnt) - (pImagAnt * pImagAnt)) + X;
        pImag = (2.0 * pRealAnt * pImagAnt) + Y;
        i++;
    }
    
    // Assign color value based on iteration count
    coloresd[(fila * img_width) + columna] = (i == maxIteracionesd) ? 0 : i;
}