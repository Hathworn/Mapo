#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Ensure only one thread modifies accuracy
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}