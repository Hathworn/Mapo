#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

struct timeval startTime, stopTime;
int started = 0;

__global__ void init(int *arr, int sqroot, int limit) {
    int c;
    for(c = 2; c <= sqroot; c++) {
        if(arr[c] == 0) {

            int tid = c + 1 + threadIdx.x + (blockIdx.x * blockDim.x);
            
            // Unrolling loop for more efficient execution
            int stride = blockDim.x * gridDim.x;
            while (tid < limit) {
                if (tid % c == 0) {
                    arr[tid] = 1;
                }
                tid += stride; // move to next block stride position
            }
        }
    }
}