#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(int *arr, int sqroot, int limit) {
    int c;
    for(c = 2; c <= sqroot; c++) {
        if(arr[c] == 0) {
            // Calculate thread's global index
            int tid = c + 1 + threadIdx.x + (blockIdx.x * blockDim.x);
            
            // Ensure within limit and mark as 1 if divisible by c
            for (int m = tid; m < limit; m += blockDim.x * gridDim.x) {
                if (m % c == 0) {
                    arr[m] = 1;
                }
            }
        }
    }
}