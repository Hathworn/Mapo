#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(int *arr, int sqroot, int limit) {
    // Compute a unique index for each thread
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Parallelize outer loop by assigning threads per value of c
    for (int c = 2; c <= sqroot; c++) {
        if (arr[c] == 0) {
            int m = c + 1 + tid;
            if (m < limit && m % c == 0) {
                arr[m] = 1;
            }
        }
    }
}