#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatMulKernel(float *Md, float *Nd, float *Pd, int width)
{
    // Global row and column index of the output matrix
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread is within matrix bounds
    if (row < width && col < width) {
        float Pvalue = 0;

        // Efficient coalesced accesses for matrix multiplication
        for (int k = 0; k < width; ++k) {
            float Melement = Md[row * width + k];
            float Nelement = Nd[k * width + col];
            Pvalue += Melement * Nelement;
        }

        // Write Pvalue to result matrix
        Pd[row * width + col] = Pvalue;
    }
}