#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatMulKernel(float *Md, float *Nd, float *Pd, int width)
{
    // Calculate thread row and column within matrix
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds to prevent out-of-bounds memory access
    if (row < width && col < width) {
        // Each thread computes one element of P
        float Pvalue = 0;

        // Multiply M and N using a loop with direct accesses
        for (int k = 0; k < width; ++k) {
            float Melement = Md[row * width + k];
            float Nelement = Nd[k * width + col];
            Pvalue += Melement * Nelement;
        }

        // Write Pvalue to device memory after computation
        Pd[row * width + col] = Pvalue;
    }
}