#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcDenseBarckwardNabraBGPU( float *dz_in, float *dB, int batch_size, int out_size_x ){
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < out_size_x) {
        float sum = 0.0f; // Use a local variable to accumulate
        for (int b = 0; b < batch_size; ++b) {
            sum += dz_in[b * out_size_x + id];
        }
        dB[id] = sum; // Write final sum back to global memory once
    }
}