#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float activator_derivative(float x) {
    float sig = 1.0f / (1.0f + exp(-x));
    return sig * (1 - sig);
}

__global__ void calcSigmoidBackwardGPU(float *dz_next_layer, float *dz_in, float *dz, float *in, int elements) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation

    if (id < elements) {
        float x = dz_in[id] += dz_next_layer[id]; // Fetch and accumulate in a single operation
        float sig = 1.0f / (1.0f + exp(-x));
        dz[id] += (sig * (1 - sig)) * dz_in[id]; // Perform the operation using local variables
    }
}