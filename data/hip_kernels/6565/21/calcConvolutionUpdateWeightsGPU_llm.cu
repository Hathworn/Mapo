#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcConvolutionUpdateWeightsGPU(float *filters, float *filter_grads, int in_size_z, int number_filters, int kernel_size, float momentum, float decay, float learning_rate, int elements) {
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    
    // Ensure thread operates within bounds
    if (id < elements) {
        int temp_id = id;
        int i = temp_id % kernel_size;
        temp_id /= kernel_size;
        int j = temp_id % kernel_size;
        temp_id /= kernel_size;
        int z = temp_id % in_size_z;
        temp_id /= in_size_z;
        int filter = temp_id;

        // Precompute constants for array offsets
        int filter_size = kernel_size * kernel_size * in_size_z;
        int filter_grad_index = (filter * filter_size + z * (kernel_size * kernel_size) + j * kernel_size + i) * 2;
        
        // Retrieve gradients and compute the momentum update
        float grad = filter_grads[filter_grad_index];
        float grad_prev = filter_grads[filter_grad_index + 1];
        float m = grad + grad_prev * momentum;
        filter_grads[filter_grad_index + 1] = m;

        // Update weight with learning rate and decay
        float w = filters[id];
        w -= learning_rate * (m + decay * w);
        filters[id] = w;
    }
}