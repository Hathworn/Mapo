#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcLeakyReluBackwardGPU(float *dz_next_layer, float *dz_in, float *dz, float *in, int elements)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplify thread index calculation

    if (id < elements) {
        float grad_factor = (in[id] < 0) ? 0.01f : 1.0f; // Pre-compute gradient factor
        dz_in[id] += dz_next_layer[id];
        dz[id] += grad_factor * dz_in[id]; // Use pre-computed gradient factor
    }
}