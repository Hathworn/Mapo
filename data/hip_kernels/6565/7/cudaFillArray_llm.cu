#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaFillArray(float *gpu_array, float val, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Combine blockIdx.x and threadIdx.x for 1D grid
    int stride = gridDim.x * blockDim.x; // Calculate stride for each block

    for (; i < N; i += stride) // Use loop to handle larger arrays
    {
        gpu_array[i] = val;
    }
}