#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcSoftmaxSumForwardGPU(float *array, float *out, float *max, float *sum, int *mutex, int batch_size, int in_size_x, unsigned n)
{
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int stride = gridDim.x * blockDim.x;

    // Use dynamic shared memory allocation
    extern __shared__ float cache[];

    float temp = 0.0;

    // Optimize by removing the offset variable and using stride with index directly
    for (unsigned int i = index; i < n; i += stride) {
        float v = exp(array[i] - max[blockIdx.x]); // Read max with blockIdx.x
        out[i] = v;
        temp += v;
    }

    // Store to cache using the available index
    cache[threadIdx.x] = temp;

    __syncthreads();

    // Perform reduction using shared memory with stride
    for (unsigned int i = blockDim.x / 2; i > 0; i /= 2) {
        if (threadIdx.x < i) {
            cache[threadIdx.x] += cache[threadIdx.x + i];
        }
        __syncthreads();
    }

    // Use atomic operations for writing the result to global memory
    if (threadIdx.x == 0) {
        while (atomicCAS(mutex, 0, 1) != 0); // Lock using atomic
        sum[blockIdx.x] += cache[0];         // Write the sum result of the block
        atomicExch(mutex, 0);                // Unlock using atomic
    }
}