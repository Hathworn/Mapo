#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int Rand(unsigned int randx)
{
    randx = randx*1103515245+12345;
    return randx&2147483647;
}
__global__ void setRandom(float *gpu_array, int N, int maxval)
{
    // Calculate flattened thread index in grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride across entire grid

    // Use stride to allow each thread to process multiple elements
    for (int id = idx; id < N; id += stride) {
        gpu_array[id] = 1.0f / maxval * Rand(id) / float(RAND_MAX);
    }
}