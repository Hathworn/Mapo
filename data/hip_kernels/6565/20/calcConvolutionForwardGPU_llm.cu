#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcConvolutionForwardGPU(float *out, float *padded_in, float *filters, int padded_in_size_x, int padded_in_size_y, int padded_in_size_z, int batch_size, int out_size_x, int out_size_y, int out_size_z, int kernel_size, int stride, int filter_size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread indexing
    int id_out = id;

    if (id_out < batch_size * out_size_x * out_size_y * out_size_z) {
        int x = id_out % out_size_x;
        int y = (id_out / out_size_x) % out_size_y;
        int filter = (id_out / (out_size_x * out_size_y)) % out_size_z;
        int b = id_out / (out_size_x * out_size_y * out_size_z);

        int mapped_x = x * stride;
        int mapped_y = y * stride;

        float sum = 0.0f;
        for (int z = 0; z < padded_in_size_z; ++z) {
            for (int j = 0; j < kernel_size; ++j) {
                for (int i = 0; i < kernel_size; ++i) {

                    int padded_in_index = b * (padded_in_size_x * padded_in_size_y * padded_in_size_z)
                                          + z * (padded_in_size_x * padded_in_size_y)
                                          + (mapped_y + j) * padded_in_size_x + (mapped_x + i);
                    int filter_index = z * (kernel_size * kernel_size) + j * kernel_size + i;

                    sum += filters[filter * filter_size + filter_index] * padded_in[padded_in_index];
                }
            }
        }
        out[id_out] = sum;
    }
}