#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcSoftmaxMaxForwardGPU(float *array, float *max, int *mutex, int batch_size, int in_size_x, unsigned n)
{
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int stride = gridDim.x * blockDim.x;
    
    extern __shared__ float cache[];

    float temp = -1.0;
    while (index + offset < n) {
        temp = fmaxf(temp, array[index + offset]);
        offset += stride;
    }

    cache[threadIdx.x] = temp; // Store results in shared memory by thread index
    __syncthreads();

    for (unsigned int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (threadIdx.x < i) {
            cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x + i]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        while (atomicCAS(mutex, 0, 1) != 0); // Lock
        max[blockIdx.x] = fmaxf(max[blockIdx.x], cache[0]); // Reduce results into global max
        atomicExch(mutex, 0); // Unlock
    }
}