#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "calcSoftmaxMaxForwardGPU.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *array = NULL;
hipMalloc(&array, XSIZE*YSIZE);
float *max = NULL;
hipMalloc(&max, XSIZE*YSIZE);
int *mutex = NULL;
hipMalloc(&mutex, XSIZE*YSIZE);
int batch_size = XSIZE*YSIZE;
int in_size_x = XSIZE*YSIZE;
unsigned n = XSIZE*YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
calcSoftmaxMaxForwardGPU<<<gridBlock, threadBlock>>>(array,max,mutex,batch_size,in_size_x,n);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
calcSoftmaxMaxForwardGPU<<<gridBlock, threadBlock>>>(array,max,mutex,batch_size,in_size_x,n);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
calcSoftmaxMaxForwardGPU<<<gridBlock, threadBlock>>>(array,max,mutex,batch_size,in_size_x,n);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}