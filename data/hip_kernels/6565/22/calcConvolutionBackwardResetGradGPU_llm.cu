#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcConvolutionBackwardResetGradGPU(float *filter_grads, int in_size_z, int kernel_size, int filter_size, int elements)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    
    if (id < elements) {
        // Calculate indices using integer division and modulus
        int i = id % kernel_size; // index within the kernel
        id /= kernel_size;
        int j = id % kernel_size; // second index within the kernel
        id /= kernel_size;
        int z = id % in_size_z; // depth index
        id /= in_size_z;
        int filter = id; // filter index

        // Update to single assignment to avoid redundant calculations
        filter_grads[(filter * in_size_z * kernel_size * kernel_size + z * kernel_size * kernel_size + j * kernel_size + i) * 2] = 0.0f; // Reset grad
    }
}