#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcDenseForwardGPU(float *in, float *out, float *weights, float *biases, int batch_size, int in_size_x, int in_size_y, int in_size_z, int out_size_x, int out_size_y, int out_size_z)
{
    // Compute flattened global thread index
    int id_out = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds for output
    if (id_out < batch_size * out_size_x * out_size_y * out_size_z) {
        int id = id_out;
        int n = id % out_size_x;
        id /= out_size_x;
        id /= out_size_y;
        id /= out_size_z;
        int b = id;

        int w_size_x = in_size_x * in_size_y * in_size_z;
        float sum = 0.0f;

        // Unroll the loop for better performance
        int in_base = b * w_size_x;
        for (int k = 0; k < in_size_z; ++k) {
            for (int j = 0; j < in_size_y; ++j) {
                int offset = (k * in_size_y + j) * in_size_x;
                for (int i = 0; i < in_size_x; ++i) {
                    int m = offset + i;
                    int w_index = n * w_size_x + m;
                    int in_index = in_base + m;

                    // Accumulate dot product
                    sum += in[in_index] * weights[w_index];
                }
            }
        }
        
        // Add bias and store result
        int bias_index = n;
        out[id_out] = sum + biases[bias_index];
    }
}