#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcReluForwardGPU(float *in, float *out, int elements)
{
    // Calculate unique thread index using built-in variables
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread is within bounds
    if(id < elements) {
        // Use max function to simplify ReLU operation
        out[id] = fmaxf(in[id], 0.0f);
    }
}