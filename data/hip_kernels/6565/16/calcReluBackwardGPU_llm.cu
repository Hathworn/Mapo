#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcReluBackwardGPU(float *dz_next_layer, float *dz_in, float *dz, float *in, int elements)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified ID calculation
    if (id < elements) {
        float dz_next = dz_next_layer[id]; // Cache read values to reduce global memory access
        float input = in[id];               // Cache read values to reduce global memory access

        dz_in[id] += dz_next;
        dz[id] += (input < 0) ? 0.0f : dz_in[id]; // Simplify ternary operation with 0.0f
    }
}