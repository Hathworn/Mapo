#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcConvolutionForwardPaddedInGPU(float *in, float *padded_in, int batch_size, int in_size_x, int in_size_y, int in_size_z, int padding) 
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified ID calculation

    if (id < batch_size * in_size_x * in_size_y * in_size_z) {
        int temp_id = id;  // Use a temporary variable for calculations
        int x = temp_id % in_size_x;
        temp_id /= in_size_x;
        int y = temp_id % in_size_y;
        temp_id /= in_size_y;
        int z = temp_id % in_size_z;
        temp_id /= in_size_z;
        int b = temp_id;

        // Pre-calculate padding offset to reduce per-thread computation
        int pad_x_size = in_size_x + 2 * padding;
        int pad_y_size = in_size_y + 2 * padding;

        int pad_index = b * (in_size_z * pad_x_size * pad_y_size) +
                        z * (pad_x_size * pad_y_size) +
                        (y + padding) * pad_x_size +
                        (x + padding);

        padded_in[pad_index] = in[id];  // Use original id for direct access
    }
}