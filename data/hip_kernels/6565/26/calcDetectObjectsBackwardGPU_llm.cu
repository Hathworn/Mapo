#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float activator_derivative(float x)
{
    float sig = 1.0f / (1.0f + exp(-x));
    return sig * (1 - sig);
}

__global__ void calcDetectObjectsBackwardGPU(float *dz_in, float *dz, float *in, int batch_size, int in_size_x, int in_size_y, int in_size_z, int max_bounding_boxes, int max_classes)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation

    int index_stride = in_size_x * in_size_y * in_size_z;

    if (id < batch_size * index_stride / (4 + max_classes)) { // Ensure id is within bounds
        for (int i = 0; i < max_bounding_boxes; i += (4 + max_classes)) {
            int index = id * index_stride + i;

            dz[index] = activator_derivative(in[index]) * dz_in[index]; // x: sigmoid derivative * grads
            dz[index + 1] = activator_derivative(in[index + 1]) * dz_in[index + 1]; // y: sigmoid derivative * grads
            dz[index + 2] = exp(in[index + 2]) * dz_in[index + 2]; // w: exp * grads
            dz[index + 3] = exp(in[index + 3]) * dz_in[index + 3]; // h: exp * grads

            for (int c = 0; c < max_classes; ++c) {
                int index2 = index + 4 + c;
                dz[index2] = activator_derivative(in[index2]) * dz_in[index2]; // id: sigmoid derivative * grads
            }
        }
    }
}