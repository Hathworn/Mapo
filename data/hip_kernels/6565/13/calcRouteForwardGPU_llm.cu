#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcRouteForwardGPU(float *in, float *out, int in_size_x, int in_size_y, int in_size_z, int z_offset, int elements)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < elements) {
        int id_in = id;

        // Precompute multiplication factors
        int area = in_size_x * in_size_y;
        int volume = area * in_size_z;

        // Optimize division and modulus operations
        int x = id % in_size_x;
        int rem = id / in_size_x;
        int y = rem % in_size_y;
        rem /= in_size_y;
        int z = rem % in_size_z;
        int b = rem / in_size_z;

        int id_out = b * volume + (z + z_offset) * area + y * in_size_x + x;
        out[id_out] = in[id_in];
    }
}