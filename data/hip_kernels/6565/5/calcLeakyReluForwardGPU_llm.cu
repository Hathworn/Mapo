#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcLeakyReluForwardGPU(float *in, float *out, int elements)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified id calculation for 1D grid

    if(id < elements) {
        float v = in[id];
        out[id] = (v < 0) ? 0.01 : v; // Use ternary operator for conditional assignment
    }
}

/* original
for( unsigned i = 0; i < data_size; ++i ){
float v = in.data[i];
if ( v < 0 ){
v = 0.01;
}
out.data[i] = v;
}
*/