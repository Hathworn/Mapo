#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcSigmoidForwardGPU(float *in, float *out, int elements)
{
    // Compute the global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Process subsequent elements using stride to improve performance
    int stride = blockDim.x * gridDim.x;

    for (; id < elements; id += stride) {
        float v = in[id];
        v = 1.0f / (1.0f + expf(-v)); // Efficient use of expf for single-precision
        out[id] = v;
    }
}