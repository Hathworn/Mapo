#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcDenseUpdateWeightsGPU(float *weights, float *biases, float *gradients, float *dW, float *dB, int batch_size, int in_size_x, int in_size_y, int in_size_z, int out_size_x, int out_size_y, int out_size_z, float learning_rate, float momentum)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < out_size_x) {
        int w_size_x = in_size_x * in_size_y * in_size_z;

        // Use shared memory to reduce global memory access
        extern __shared__ float shared_dW[];
        
        for (int h = threadIdx.x; h < w_size_x; h += blockDim.x) {
            int index = h * out_size_x + id;
            shared_dW[h] = dW[index];
        }
        __syncthreads();
        
        for (int h = 0; h < w_size_x; ++h) {
            int index = h * out_size_x + id;
            weights[index] -= learning_rate * shared_dW[h];
        }
        
        biases[id] -= learning_rate * dB[id];
        
        // Unroll the loop for pipeline efficiency
        for (int b = 0; b < batch_size; b += 2) {
            int index1 = (b * out_size_x + id) * 2;
            int index2 = index1 + 2 * out_size_x;
            gradients[index1 + 1] = gradients[index1] + gradients[index1 + 1] * momentum;

            if (b + 1 < batch_size) {
                gradients[index2 + 1] = gradients[index2] + gradients[index2 + 1] * momentum;
            }
        }
    }
}