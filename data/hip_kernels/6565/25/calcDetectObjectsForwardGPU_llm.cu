#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void calcDetectObjectsForwardGPU(float *in, float *out, int in_size_x, int in_size_y, int in_size_z, int max_bounding_boxes, int max_classes)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int input_stride = in_size_x * in_size_y * in_size_z; // Precompute stride

    // Loop through bounding boxes
    for (int i = 0; i < max_bounding_boxes; i += (4 + max_classes)) {
        int index = id * input_stride + i;

        // Use shared memory for common calculations
        float common_in0 = in[index];
        float common_in1 = in[index + 1];
        
        // Directly store sigmoid and exp calculations to output
        out[index] = 1.0f / (1.0f + exp(-common_in0)); // x: sigmoid
        out[index + 1] = 1.0f / (1.0f + exp(-common_in1)); // y: sigmoid
        out[index + 2] = exp(in[index + 2]); // w: exp
        out[index + 3] = exp(in[index + 3]); // h: exp

        // Loop through classes and compute sigmoid
        #pragma unroll
        for (int c = 0; c < max_classes; ++c) {
            int index2 = id * input_stride + i + 4 + c;
            out[index2] = 1.0f / (1.0f + exp(-in[index2])); // id: sigmoid
        }
    }
}