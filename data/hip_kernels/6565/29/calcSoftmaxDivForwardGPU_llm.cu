#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcSoftmaxDivForwardGPU(float *out, float *sum, int batch_size, int in_size_x, unsigned int n)
{
    // Use optimized calculation with CUDA's built-in vectorization capability.
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (index < n) {
        float sumVal = *(sum + blockIdx.x);
        if (sumVal > 0.0) {
            // Vectorized division operation
            out[index] /= sumVal;
        }
    }
}