#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcSoftmaxBackwardGPU(float *dz_next_layer, float *dz_in, float *dz, unsigned int n)
{
    // Calculate global thread index
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check within bounds
    if (index < n) {
        // Perform computation
        dz_in[index] += dz_next_layer[index];
        dz[index] += dz_in[index];
    }
}