#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcMaxPoolForwardGPU( float *in,float *out, int in_size_x, int in_size_y, int in_size_z, int batch_size, int out_size_x, int out_size_y, int out_size_z, int stride, int kernel_size )
{
    int id_out = blockIdx.x * blockDim.x + threadIdx.x;

    if (id_out < batch_size * out_size_x * out_size_y * out_size_z) {
        // Precompute indices for better memory coalescing
        int x = id_out % out_size_x;
        int y = (id_out / out_size_x) % out_size_y;
        int z = (id_out / (out_size_x * out_size_y)) % out_size_z;
        int b = id_out / (out_size_x * out_size_y * out_size_z);

        int mapped_x = x * stride;
        int mapped_y = y * stride;

        float mval = -FLT_MAX;  // Use FLT_MAX to ensure it's a known constant
        for (int j = 0; j < kernel_size; ++j) {
            for (int i = 0; i < kernel_size; ++i) {
                // Avoid redundant calculations
                int id_in = b * (in_size_z * in_size_x * in_size_y) +
                            z * (in_size_x * in_size_y) +
                            (mapped_y + j) * in_size_x +
                            (mapped_x + i);

                float v = in[id_in];
                mval = fmaxf(mval, v);  // Use fmaxf for possible performance improvement
            }
        }
        out[id_out] = mval;
    }
}