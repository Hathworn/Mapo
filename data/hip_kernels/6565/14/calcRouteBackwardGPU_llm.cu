#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcRouteBackwardGPU( float *dz_in, float *dz, int in_size_x, int in_size_y, int in_size_z, int z_offset, int elements )
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for efficient global memory access
    
    for (; id < elements; id += stride) { // Loop over elements with stride
        int temp_id = id; // Use a temporary variable to preserve original id for indexing
        int x = temp_id % in_size_x;
        temp_id /= in_size_x;
        int y = temp_id % in_size_y;
        temp_id /= in_size_y;
        int z = temp_id % in_size_z;
        temp_id /= in_size_z;
        int b = temp_id;

        int id_in = b * (in_size_z * in_size_x * in_size_y) + (z + z_offset) * (in_size_x * in_size_y) + y * in_size_x + x;
        atomicAdd(&dz[id], dz_in[id_in]); // Use atomicAdd to prevent race conditions
    }
}