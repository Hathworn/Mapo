#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calcDenseBackwardGPU( float *dz_in, float *dz, float *in, float *weights, float *biases, float *gradients, float *dW, float *dB, int batch_size, int in_size_x, int in_size_y, int in_size_z, int out_size_x, int out_size_y, int out_size_z, float momentum, float decay )
{
    // Calculate global thread ID
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int id_out = id;
    if( id_out < batch_size * in_size_x * in_size_y * in_size_z * out_size_x ) {
        
        // Calculate input and output indices
        int id_in  = id / out_size_x;
        int n = id % out_size_x;
        id /= out_size_x;
        int i = id % in_size_x;
        id /= in_size_x;
        int j = id % in_size_y;
        id /= in_size_y;
        int z = id % in_size_z;
        int b = id / in_size_z;

        int w_size_x = in_size_x * in_size_y * in_size_z;
        int m = z * (in_size_x * in_size_y) + j * (in_size_x) + i;

        // Load and calculate without a loop for n
        float dzin = dz_in[b * (out_size_x * out_size_y * out_size_z) + n];
        int w_index = n * w_size_x + m;
        float w = weights[w_index];

        gradients[(n * batch_size + b) * 2] = dzin;
        atomicAdd(&dz[id_in], dzin * w);  // Use atomicAdd to prevent race conditions
        atomicAdd(&dW[w_index], in[id_in] * (gradients[(n * batch_size + b) * 2] + gradients[(n * batch_size + b) * 2 + 1] * momentum) + (decay * w));
    }
}