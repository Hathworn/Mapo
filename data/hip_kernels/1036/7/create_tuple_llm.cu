#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
double* x;
int numSMs;

__global__ void create_tuple(double* devx, int* pos_x, int* pos_y, double* value, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    if (index < N) {
        pos_x[index] = index;
        pos_y[index] = index;
        value[index] = 2 * cosf(2 * devx[index]);
    }
    else if (index == N) {
        pos_x[index] = N;  // Ensure boundary condition for last element
    }
}