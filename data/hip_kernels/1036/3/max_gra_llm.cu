#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

double* x;
int numSMs;

__device__ __host__ inline double Max(double x, double y) {
    x = fabs(x);
    y = fabs(y);
    return x > y ? x : y;
}

__global__ void max_gra(double* gra, double* max) {
    extern __shared__ double sharedGra[];  // Use shared memory
    
    int index = threadIdx.x;
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    sharedGra[index] = gra[globalIndex];
    __syncthreads();

    // Perform reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (index < stride) {
            sharedGra[index] = Max(sharedGra[index], sharedGra[index + stride]);
        }
        __syncthreads();
    }

    // Write result for this block to max
    if (index == 0) {
        atomicMax((unsigned long long int*)max, __double_as_longlong(sharedGra[0]));
    }
}