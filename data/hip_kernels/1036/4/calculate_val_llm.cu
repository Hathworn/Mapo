#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

double* x;
int numSMs;

__device__ double sqr(double x) {
    return x * x;
}

__global__ void calculate_val(double* devx, double* val, int size) {
    // Calculate unique thread index across the entire grid
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a stride loop to cover all elements with this thread
    int stride = blockDim.x * gridDim.x;
    while (index < size) {
        int pre = index - 1; 
        if (pre < 0) pre += size;
        int next = index + 1;
        if (next >= size) next -= size;
        val[index] = sqr(sin(devx[pre] * devx[index])) * sqr(sin(devx[next] * devx[index]));
        index += stride; // Increment by stride for the next iteration
    }
}