#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

double* x;
int numSMs;

__global__ void minus_gra(double* gra, int size) {
    // Calculate global index for the current thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Process the work in a coalesced manner
    while (index < size) {
        gra[index] = -gra[index];  // Directly negate the value
        index += blockDim.x * gridDim.x;  // Move to the next step
    }
}