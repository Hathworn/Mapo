#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
double* x;
int numSMs;

__global__ void printdd(int* pos, int size) {
    // Use each thread to access a specific element in parallel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        printf("csr %d\n", pos[idx]);
    }
}