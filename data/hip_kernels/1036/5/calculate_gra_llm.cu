#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

double* x;
int numSMs;

__device__ double sqr(double x) {
    return x * x;
}

__global__ void calculate_gra(double* devx, double* gra, int size) {
    // Use shared memory for faster access and reduce global memory interactions
    extern __shared__ double shared_devx[];
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index < size) {
        shared_devx[threadIdx.x] = devx[index];
        __syncthreads(); // Ensure all threads have written to shared memory
        
        int pre = index - 1;
        if (pre < 0) pre += size;
        int next = index + 1;
        if (next >= size) next -= size;
        
        // Access through shared memory if within bounds
        double devx_pre = (threadIdx.x > 0) ? shared_devx[threadIdx.x - 1] : devx[pre];
        double devx_next = (threadIdx.x < blockDim.x - 1) ? shared_devx[threadIdx.x + 1] : devx[next];
        
        gra[index] = devx_pre * sin(2.0 * devx[index] * devx_pre) + devx_next * sin(2.0 * devx[index] * devx_next);
        printf("gra %d %d %d %f %f %f\n", pre, index, next, sqr(devx[index]), devx_pre * sin(2.0 * devx[index] * devx_pre), gra[index]);
    }
}