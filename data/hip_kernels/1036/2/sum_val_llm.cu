#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

int numSMs;

__global__ void sum_val(double* val, double* r) {
    // Using shared memory for better performance
    extern __shared__ double shared_val[];
    int index = threadIdx.x;

    // Load data into shared memory
    shared_val[index] = val[index];
    __syncthreads();

    // Perform parallel reduction in shared memory
    for (int i = 1; i < blockDim.x; i <<= 1) {
        int pair_index = index + i;
        if (index % (i << 1) == 0 && pair_index < blockDim.x) {
            shared_val[index] += shared_val[pair_index];
        }
        __syncthreads();
    }

    // Write result to global memory
    if (index == 0) {
        r[0] = shared_val[0];
    }
}