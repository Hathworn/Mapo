#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMul_kernel(float * A, float * B, float * C, int N)
{
    const int TILE_WIDTH = 16;
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    float tmpSum = 0.0f; // Temporary sum for the element

    for (int t = 0; t < (N - 1) / TILE_WIDTH + 1; ++t) {
        // Collaborative loading of data into shared memory
        if (row < N && t * TILE_WIDTH + threadIdx.x < N)
            tileA[threadIdx.y][threadIdx.x] = A[row * N + t * TILE_WIDTH + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < N && t * TILE_WIDTH + threadIdx.y < N)
            tileB[threadIdx.y][threadIdx.x] = B[(t * TILE_WIDTH + threadIdx.y) * N + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads(); // Ensure all threads have loaded their data

        // Perform computation on the tile
        for (int i = 0; i < TILE_WIDTH; ++i) {
            tmpSum += tileA[threadIdx.y][i] * tileB[i][threadIdx.x];
        }
        __syncthreads(); // Ensure all threads have finished computing on the current tile
    }

    // Write the result back to the global memory
    if (row < N && col < N)
        C[row * N + col] = tmpSum;
}