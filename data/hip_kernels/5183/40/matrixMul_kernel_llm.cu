#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Macro for tile size
#define TILE_SIZE 16

__global__ void matrixMul_kernel(float *A, float *B, float *C, int N)
{
    // Shared memory for tiles
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    int ROW = blockIdx.y * TILE_SIZE + threadIdx.y;
    int COL = blockIdx.x * TILE_SIZE + threadIdx.x;
    float tmpSum = 0.0f;

    // Loop over tiles
    for (int m = 0; m < N / TILE_SIZE; m++)
    {
        // Load data into shared memory
        if (ROW < N && m * TILE_SIZE + threadIdx.x < N)
            tileA[threadIdx.y][threadIdx.x] = A[ROW * N + m * TILE_SIZE + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        if (COL < N && m * TILE_SIZE + threadIdx.y < N)
            tileB[threadIdx.y][threadIdx.x] = B[(m * TILE_SIZE + threadIdx.y) * N + COL];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        // Perform computation for tiles
        for (int k = 0; k < TILE_SIZE; k++)
        {
            tmpSum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }
        
        __syncthreads();
    }

    // Write the result to global memory
    if (ROW < N && COL < N)
    {
        C[ROW * N + COL] = tmpSum;
    }
}