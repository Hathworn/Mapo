#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMul_kernel(float * A, float * B, float * C, int N)
{
    // Calculate row and column index for each thread
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize local sum accumulation
    float tmpSum = 0;

    // Check bounds to ensure we are within matrix dimensions
    if (ROW < N && COL < N)
    {
        // Unroll the loop where possible to improve performance
        for (int i = 0; i < N; i += 4)
        {
            // Perform operation in chunks to utilize instruction level parallelism
            tmpSum += A[ROW * N + i] * B[i * N + COL];
            if (i + 1 < N) tmpSum += A[ROW * N + i + 1] * B[(i + 1) * N + COL];
            if (i + 2 < N) tmpSum += A[ROW * N + i + 2] * B[(i + 2) * N + COL];
            if (i + 3 < N) tmpSum += A[ROW * N + i + 3] * B[(i + 3) * N + COL];
        }
    }
    // Store the computed value in the result matrix
    if (ROW < N && COL < N)
        C[ROW * N + COL] = tmpSum;
}