#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access by using shared memory, and handle uncoalesced access
__global__ void simple_add(float* a, float* b, int n) {
    extern __shared__ float shared_b[];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    // Load b into shared memory
    if (i < n) {
        shared_b[tid] = b[i];
        __syncthreads();

        // Perform addition using shared memory
        a[i] += shared_b[tid];
    }
}