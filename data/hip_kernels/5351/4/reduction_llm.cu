#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(int* input, int* output) {
    __shared__ int tmp[TPB];

    // Load input into shared memory
    tmp[threadIdx.x] = input[threadIdx.x + blockIdx.x * blockDim.x];
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            tmp[threadIdx.x] += tmp[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        output[blockIdx.x] = tmp[0];
    }
}