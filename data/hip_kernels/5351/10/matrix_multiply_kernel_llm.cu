#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize for memory access and computation efficiency
__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    // Use shared memory for the input vector to reduce global memory access
    extern __shared__ double shared_vector[];
    
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < dim_mn) {
        double out = 0.;
        
        // Copy the input vector to shared memory
        if (threadIdx.x < dim_mn) {
            shared_vector[threadIdx.x] = vector_in[threadIdx.x];
        }
        __syncthreads(); // Ensure all threads have loaded the vector
        
        // Perform the matrix-vector multiplication with shared memory access
        for (long j = 0; j < dim_mn; j++) {
            out += matrix[i * dim_mn + j] * shared_vector[j];
        }
        
        vector_out[i] = out;
    }
}