#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bigstencil(int* in, int* out) {
    // Calculate the global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for better memory access efficiency
    __shared__ int shared_in[256];  // Adjust size as needed based on blockDim.x
    shared_in[threadIdx.x] = in[i];

    __syncthreads();  // Ensure all threads have loaded data into shared memory

    // Perform computation
    out[i] = shared_in[threadIdx.x] + 2;
}