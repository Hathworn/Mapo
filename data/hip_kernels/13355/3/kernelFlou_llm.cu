#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFlou(unsigned char * ptr, unsigned int * debug)
{
    __shared__ char sum[4];

    // Initialize shared memory to zero using a loop to reduce code redundancy
    for (int i=0; i<4; i++) {
        sum[i] = 0;
    }

    int x = blockIdx.x;
    int y = blockIdx.y;
    int cc = threadIdx.z;

    int index_ptr = (x * DIM_2 + threadIdx.x + (y * DIM_2 + threadIdx.y) * (gridDim.x * DIM_2)) * 4;
    int index_avg = (x + y * gridDim.x) * 4;

    __syncthreads();

    // Atomic addition for safe concurrent updates of shared memory
    atomicAdd(&sum[cc], ptr[index_ptr + cc] / (DIM_2 * DIM_2));

    __syncthreads();

    ptr[index_ptr + cc] = sum[cc];
    debug[index_avg + cc] = sum[cc];
}