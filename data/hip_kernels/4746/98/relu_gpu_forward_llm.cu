#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Use shared memory to optimize access
    extern __shared__ float shared_in[];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        // Load data into shared memory
        shared_in[threadIdx.x] = in[tid];
        __syncthreads();

        // Compute relu using shared memory
        out[tid] = shared_in[threadIdx.x] > 0 ? shared_in[threadIdx.x] : 0;
    }
}