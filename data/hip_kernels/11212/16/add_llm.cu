#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 1024  // Assuming a macro definition for N

__global__ void add(int *a, int *b, int *c) {
    // Get the thread id with consideration of block and thread dimensions
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform addition only if within bounds
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}