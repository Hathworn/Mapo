#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add( int *a, int *b, int *c ) {
    // Calculate global thread ID
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    // Ensure tid within bounds
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}