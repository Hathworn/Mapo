#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel( unsigned char *ptr, int ticks ) {
    // Use shared memory for intermediate calculations
    __shared__ float sh_fx;
    __shared__ float sh_fy;
    __shared__ float sh_d;
    
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // Calculate pixel position relative to center
    fx = x - DIM/2;
    fy = y - DIM/2;

    // Preload shared memory with calculated values
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        sh_fx = fx;
        sh_fy = fy;
        sh_d = sqrtf( sh_fx * sh_fx + sh_fy * sh_fy );
    }
    __syncthreads();
    
    // Calculate grey level using shared memory value
    unsigned char grey = (unsigned char)(128.0f + 127.0f *
                  cos(sh_d/10.0f - ticks/7.0f) /
                  (sh_d/10.0f + 1.0f));
    ptr[offset*4 + 0] = grey;
    ptr[offset*4 + 1] = grey;
    ptr[offset*4 + 2] = grey;
    ptr[offset*4 + 3] = 255;
}