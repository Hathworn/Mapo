#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel( int *a, int *b, int *c ) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        // Use shared memory to reduce global memory accesses
        __shared__ int shared_a[256];
        __shared__ int shared_b[256];

        shared_a[threadIdx.x] = a[idx];
        shared_b[threadIdx.x] = b[idx];

        __syncthreads();

        int idx1 = (threadIdx.x + 1) % 256;
        int idx2 = (threadIdx.x + 2) % 256;
        
        float as = (shared_a[threadIdx.x] + shared_a[idx1] + shared_a[idx2]) / 3.0f;
        float bs = (shared_b[threadIdx.x] + shared_b[idx1] + shared_b[idx2]) / 3.0f;

        // Eliminate repeated divisions to improve performance
        c[idx] = (as + bs) * 0.5f;
    }
}