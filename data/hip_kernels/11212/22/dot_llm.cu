#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(float *a, float *b, float *c) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0f;
    // Process multiple elements per thread to improve performance
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        temp += a[i] * b[i];
    }

    // Set shared cache with the computed values
    cache[cacheIndex] = temp;
    
    // Synchronize threads within the block
    __syncthreads();

    // Perform reduction in the shared memory
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    // Store the result from each block
    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}