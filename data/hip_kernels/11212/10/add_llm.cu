#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c) {
    // Use shared memory for better performance on write
    __shared__ int temp;
    if (threadIdx.x == 0) {
        temp = a + b;
        *c = temp;
    }
}