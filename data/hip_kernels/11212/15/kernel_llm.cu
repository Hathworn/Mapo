#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(uchar4 *ptr, int ticks) {
    // Map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // Pre-compute constants and values used in calculations
    float center = DIM / 2.0f;
    float fx = x - center;
    float fy = y - center;
    float d = sqrtf(fx * fx + fy * fy);
    float angle = d / 10.0f - ticks / 7.0f;
    
    // Compute grey value using optimized formula
    unsigned char grey = (unsigned char)(128.0f + 127.0f * cosf(angle) / (d / 10.0f + 1.0f));
    
    // Set uchar4 values efficiently
    uchar4 pixel = make_uchar4(grey, grey, grey, 255);
    ptr[offset] = pixel;
}