#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo) {
    // Use a static shared memory size for better performance
    __shared__ unsigned int temp[256];
    
    // Initialize shared memory using single thread
    if (threadIdx.x < 256) {
        temp[threadIdx.x] = 0;
    }
    __syncthreads();
    
    // Calculate the starting index and the offset for processing
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    // Process buffer in stride pattern
    while (i < size) {
        atomicAdd(&temp[buffer[i]], 1);
        i += stride;
    }
    __syncthreads();

    // Update global histogram using data from shared memory
    if (threadIdx.x < 256) {
        atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
    }
}