#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot(int size, float *a, float *b, float *c) {
    extern __shared__ float cache[];  // Use dynamic shared memory
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0f;
    // Loop unrolling to reduce loop overhead
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        temp += a[i] * b[i];
    }

    // Store temp result in cache
    cache[cacheIndex] = temp;
    __syncthreads();

    // Parallel reduction within shared memory
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    // Store the final result of this block to the output array
    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}