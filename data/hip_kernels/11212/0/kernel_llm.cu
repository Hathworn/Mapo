#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)

__global__ void kernel(int *a, int *b, int *c) {
    // Calculate global index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Use shared memory to improve data locality and reduce global memory accesses
    extern __shared__ int shared[];
    int* sharedA = shared;
    int* sharedB = shared + blockDim.x;

    if (idx < N) {
        int localIdx = threadIdx.x;

        // Load elements to shared memory
        sharedA[localIdx] = a[idx];
        sharedB[localIdx] = b[idx];
        __syncthreads();

        // Handle edge cases within shared memory block
        int idx1 = (localIdx + 1) % blockDim.x;
        int idx2 = (localIdx + 2) % blockDim.x;

        // Compute as and bs using shared memory
        float as = (sharedA[localIdx] + sharedA[idx1] + sharedA[idx2]) / 3.0f;
        float bs = (sharedB[localIdx] + sharedB[idx1] + sharedB[idx2]) / 3.0f;

        // Store result in global memory
        c[idx] = (as + bs) / 2.0f;
    }
}