#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add( int *a, int *b, int *c ) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;  // Calculate the stride

    // Use stride to handle larger arrays efficiently
    for (int i = tid; i < N; i += stride) {
        c[i] = a[i] + b[i];  // Perform addition
    }
}