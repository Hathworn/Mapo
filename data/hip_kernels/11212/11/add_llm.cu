#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int addem( int a, int b ) {
    return a + b;
}

// Optimized kernel function
__global__ void add( int a, int b, int *c ) {
    int result = addem(a, b); // Use local register for intermediate result
    if (threadIdx.x == 0) { // Ensure only one thread writes to *c
        *c = result;
    }
}