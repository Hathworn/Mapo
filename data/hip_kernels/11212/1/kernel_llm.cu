#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        // Use shared memory to minimize global memory access
        __shared__ int shared_a[256];
        __shared__ int shared_b[256];

        int localIdx = threadIdx.x;

        // Load data into shared memory
        shared_a[localIdx] = a[idx];
        shared_b[localIdx] = b[idx];
        __syncthreads(); // Ensure all threads have written to shared memory

        // Compute indices
        int idx1 = (localIdx + 1) % 256;
        int idx2 = (localIdx + 2) % 256;

        // Perform calculations using shared memory
        float as = (shared_a[localIdx] + shared_a[idx1] + shared_a[idx2]) / 3.0f;
        float bs = (shared_b[localIdx] + shared_b[idx1] + shared_b[idx2]) / 3.0f;

        // Store result in global memory
        c[idx] = (as + bs) / 2;
    }
}