#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(int size, float *a, float *b, float *c) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    // Unroll loop for better memory access and computation
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        temp += a[i] * b[i];
    }

    // Set cache values
    cache[cacheIndex] = temp;

    // Synchronize threads in this block
    __syncthreads();

    // Perform parallel reduction
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    // Store result for this block
    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}