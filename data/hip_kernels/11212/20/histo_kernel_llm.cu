#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo) {
    // Use shared memory to reduce atomic operations in global memory
    __shared__ unsigned int local_histo[256];
    
    // Initialize shared memory histogram
    int local_index = threadIdx.x;
    if (local_index < 256) {
        local_histo[local_index] = 0;
    }
    __syncthreads();
    
    // Calculate the starting index and the offset to the next block
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    while (i < size) {
        atomicAdd(&local_histo[buffer[i]], 1);
        i += stride;
    }
    __syncthreads();
    
    // Accumulate results from shared memory to global memory
    if (local_index < 256) {
        atomicAdd(&histo[local_index], local_histo[local_index]);
    }
}