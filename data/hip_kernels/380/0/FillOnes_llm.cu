#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FillOnes(float* vec, int value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if idx is within valid range and exit early if not.
    if(idx >= value) return;

    // Directly assign the value 1.0f to the vector at idx.
    vec[idx] = 1.0f;
}