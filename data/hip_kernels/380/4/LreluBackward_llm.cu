#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LreluBackward(float* srcDiff, float* dstDiff, float* srcData, int data_size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Utilize coalesced memory access by iterating over the same data size but with strided access
    for (int i = index; i < data_size; i += blockDim.x * gridDim.x)
    {
        dstDiff[i] = srcDiff[i] * ((srcData[i] > 0) + (srcData[i] <= 0) * 0.01);
    }
}