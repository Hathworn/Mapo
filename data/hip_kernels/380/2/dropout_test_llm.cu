#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dropout_test(float* data, int size, float probability)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Optimize using strided loop and eliminate redundant multiplication
    for(int i = index; i < size; i += stride)
    {
        data[i] *= probability; // Direct in-place update
    }
}