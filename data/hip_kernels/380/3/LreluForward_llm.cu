#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LreluForward(float* srcData, float* dstData, int data_size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // Ensure each thread handles one element, removing unnecessary loop
    if(index < data_size)
    {
        dstData[index] = srcData[index] > 0 ? srcData[index] : srcData[index] * 0.01;
    }
}