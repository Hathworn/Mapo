#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dropout_train(float* data, float* outputPtr, int size, float probability)
{
    // Calculate thread index
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Use stride loop optimization
    for (int i = index; i < size; i += stride)
    {
        if (outputPtr[i] < probability)
        {
            data[i] = 0;
        }
    }
}