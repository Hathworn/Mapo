#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SoftmaxLossBackprop(const int* label, int num_labels, int batch_size, float* diffData)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < batch_size)  // Simplified condition check
    {
        const int label_value = label[idx];
        diffData[idx * num_labels + label_value] -= 1.0f;
    }
}