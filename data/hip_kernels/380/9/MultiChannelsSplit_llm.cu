#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiChannelsSplit(float* inputs, float* outputs, int outChannels, int offset, int row, int inChannels)
{
    int batchId = blockIdx.x;
    float* input = inputs + batchId * inChannels * row * row + offset;

    int blockDo  = outChannels * row * row;
    int j = blockIdx.y * blockDim.x + threadIdx.x; // Spread the work across multiple blocks

    if(j < blockDo)
    {
        int pos = batchId * outChannels * row * row;
        outputs[pos + j] = input[j];
    }
}