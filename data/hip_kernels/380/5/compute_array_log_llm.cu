#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_array_log(float* array, float* groundTruth, int size)
{
    int thread_index = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = blockDim.x * gridDim.x;

    // Loop unrolling for improved performance
    for(int i = thread_index; i < size; i += num_threads)
    {
        groundTruth[i] = logf(array[i]) * groundTruth[i];  // Use logf for floating-point
    }
}