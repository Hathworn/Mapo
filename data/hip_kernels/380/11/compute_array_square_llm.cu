#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_array_square(float* array, float* outArray, int size)
{
    // Calculate the global thread index
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use a simple stride loop to cover all elements efficiently
    for (int i = index; i < size; i += blockDim.x * gridDim.x)
    {
        outArray[i] = array[i] * array[i];
    }
}