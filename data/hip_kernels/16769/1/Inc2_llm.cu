#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc2(float *Ad, float *Bd) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx < 1024) {
        float A_val = Ad[tx]; // Load Ad[tx] to a register for faster access
        for (int i = 0; i < ITER; i++) {
            A_val += 1.0f; // Increment A_val instead of Ad[tx]
            float B_val = A_val; // Store in a temporary variable to minimize global memory accesses
            for (int j = 0; j < 256; j++) {
                Bd[tx] = B_val; // Write once to global memory
            }
        }
        Ad[tx] = A_val; // Write the updated value back to Ad[tx]
    }
}