#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc1(float *Ad, float *Bd){
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if(tx < 1 ){
        float a = Ad[tx];
        for(int i = 0; i < ITER; i++){
            a = a + 1.0f;
        }
        Ad[tx] = a; // Write back to global memory once
        float b = a;
        for(int j = 0; j < 256; j++){
            Bd[tx] = b; // Remove dependency on external loop
        }
    }
}