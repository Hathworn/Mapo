#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_valid_pos(int32_t* pos_buff, int32_t* count_buff, const int32_t entry_count) {
    const int32_t start = threadIdx.x + blockDim.x * blockIdx.x;
    const int32_t step = blockDim.x * gridDim.x;
    
    // Use shared memory to minimize global memory access for frequently accessed data
    __shared__ int32_t shared_count_buff[256]; // Assuming blockDim.x <= 256

    for (int32_t i = start; i < entry_count; i += step) {
        if (VALID_POS_FLAG == pos_buff[i]) {
            // Use shared memory to check and update counts
            if (threadIdx.x > 0) {
                shared_count_buff[threadIdx.x] = count_buff[i - 1];
            }
            __syncthreads(); // Ensure all threads have updated shared memory

            pos_buff[i] = threadIdx.x == 0 ? 0 : shared_count_buff[threadIdx.x];

            __syncthreads(); // Ensure all writes to global memory are complete
        }
    }
}