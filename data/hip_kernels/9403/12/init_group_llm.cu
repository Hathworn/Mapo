#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_group(int8_t* groups, const size_t group_count, const size_t col_count, const size_t* col_widths, const size_t* init_vals) {
    // Calculate the global thread index
    const auto thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Early exit for threads beyond group_count
    if (thread_index >= group_count) {
        return;
    }
    
    // Loop optimization: remove repeated computations
    for (size_t i = 0, offset = 0; i < col_count; offset += col_widths[i++] * group_count) {
        switch (col_widths[i]) {
            case 4:
                // Use reinterpret_cast for 4-byte case directly
                *reinterpret_cast<uint32_t*>(groups + offset + thread_index * col_widths[i]) =
                *reinterpret_cast<const uint32_t*>(init_vals + i);
                break;
            case 8:
                // Access the groups memory using calculated offset
                reinterpret_cast<size_t*>(groups + offset)[thread_index] = init_vals[i];
                break;
            default:
                // Handle other cases if necessary
                break;
        }
    }
}