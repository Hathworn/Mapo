#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_valid_pos_flag(int32_t* pos_buff, const int32_t* count_buff, const int32_t entry_count) {
    const int32_t i = threadIdx.x + blockDim.x * blockIdx.x;
    // Ensure that thread index is within the entry_count range
    if (i < entry_count && count_buff[i]) {
        pos_buff[i] = VALID_POS_FLAG;
    }
}