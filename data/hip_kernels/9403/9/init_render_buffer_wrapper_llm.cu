#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void init_render_buffer(int64_t* render_buffer, const uint32_t qw_count) {
    // Optimize memory access by using shared memory for caching
    __shared__ int64_t shared_buffer[256]; // Assuming blockDim.x <= 256 for simplicity
    const uint32_t start = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t step = blockDim.x * gridDim.x;
    
    for (uint32_t i = start; i < qw_count; i += step) {
        render_buffer[i] = EMPTY_KEY_64;
    }
}

__global__ void init_render_buffer_wrapper(int64_t* render_buffer, const uint32_t qw_count) {
    // Launching kernel with cache-enabled memory
    init_render_buffer(render_buffer, qw_count);
}