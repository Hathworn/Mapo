#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void check_if_unique(const unsigned *keys, unsigned *is_unique, size_t kSize) {
    unsigned id = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Check if within bounds
    if (id < kSize) {
        if (id == 0) {
            // Initialize first element as unique
            is_unique[0] = 1;
        } else {
            // Set unique flag based on comparison with previous element
            is_unique[id] = (keys[id] != keys[id - 1]);
        }
    }
}