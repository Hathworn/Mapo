#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prepare_indices(const unsigned num_keys, unsigned *data) {
    // Improve index calculation efficiency
    unsigned idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned index = idx_x + idx_y * gridDim.x * blockDim.x;

    // Check if index is within bounds
    if (index < num_keys) {
        data[index] = index;
    }
}