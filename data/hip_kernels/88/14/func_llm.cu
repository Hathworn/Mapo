#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void func (char* stringInput, int stringSize, int* integerInput, char* dummySpace) {
    // Use thread index to calculate the position in the output array
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int totalSize = stringSize + sizeof(int);

    // Ensure index is within bounds
    if (idx < totalSize) {
        if (idx < stringSize) {
            // Copy string input to dummySpace
            dummySpace[idx] = stringInput[idx];
        } else {
            // Copy integer input to dummySpace
            dummySpace[idx] = ((char*)integerInput)[idx - stringSize];
        }
    }
}