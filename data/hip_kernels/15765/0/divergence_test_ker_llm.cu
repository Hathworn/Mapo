#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void divergence_test_ker()
{
    // Use shared memory for reduced divergence
    __shared__ int tid_even[1024]; 
    __shared__ int tid_odd[1024];

    // Store thread index in appropriate array
    if (threadIdx.x % 2 == 0) {
        tid_even[threadIdx.x / 2] = threadIdx.x;
    } else {
        tid_odd[threadIdx.x / 2] = threadIdx.x;
    }

    __syncthreads(); // Synchronize before printing to avoid race conditions

    // Even thread printing
    if (threadIdx.x % 2 == 0) {
        printf("threadIdx.x %d : This is an even thread.\n", tid_even[threadIdx.x / 2]);
    }
    // Odd thread printing
    else {
        printf("threadIdx.x %d : This is an odd thread.\n", tid_odd[threadIdx.x / 2]);
    }
}