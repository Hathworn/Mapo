#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float rowcol_dot(float *matrix_a, float *matrix_b, int row, int col, int N)
{
    float val = 0;
    for (int k = 0; k < N; k++)
    {
        val += matrix_a[row * N + k] * matrix_b[col + k * N];
    }
    return val;
}

__global__ void matrix_mult_ker(float *matrix_a, float *matrix_b, float *output_matrix, int N)
{
    // Compute row and column indices within the grid.
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if within bounds before computation.
    if (row < N && col < N)
    {
        output_matrix[col + row * N] = rowcol_dot(matrix_a, matrix_b, row, col, N);
    }
}