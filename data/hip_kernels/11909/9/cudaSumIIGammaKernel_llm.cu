#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

static unsigned int GRID_SIZE_N;
static unsigned int GRID_SIZE_4N;
static unsigned int MAX_STATE_VALUE;

__global__ static void cudaSumIIGammaKernel(double *x1, double *x2, double *sumtable, int limit) {
    // Calculate unique thread index
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Return if index is beyond the limit
    if (n >= limit) return;

    // Compute indices for accessing input and result arrays
    const int i = n / 4, l = n % 4;
    double *left = &(x1[16 * i + l * 4]);
    double *right = &(x2[16 * i + l * 4]);
    double *sum = &(sumtable[i * 16 + l * 4]);

    // Unrolled loop to leverage instruction-level parallelism
    #pragma unroll
    for (int k = 0; k < 4; k++) {
        sum[k] = left[k] * right[k];
    }
}