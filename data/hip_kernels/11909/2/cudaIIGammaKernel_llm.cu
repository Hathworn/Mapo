#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

static unsigned int GRID_SIZE_N;
static unsigned int GRID_SIZE_4N;
static unsigned int MAX_STATE_VALUE;

__global__ static void cudaIIGammaKernel(double *extEV, double *x1, double *x2, double *x3, double *left, double *right) {
    __shared__ volatile double al[16], ar[16], x1px2[16];
    const int tid = threadIdx.z * 16 + threadIdx.y * 4 + threadIdx.x;
    const int offset = 16 * blockIdx.x + threadIdx.z * 4;
    const int lane = threadIdx.x;

    double local_al = x1[offset + lane] * left[tid];
    double local_ar = x2[offset + lane] * right[tid];

    // Reduce values within each warp
    local_al += __shfl_xor_sync(0xFFFFFFFF, local_al, 1);
    local_ar += __shfl_xor_sync(0xFFFFFFFF, local_ar, 1);

    if (lane == 0) {
        al[threadIdx.z * 4 + threadIdx.y] = local_al;
        ar[threadIdx.z * 4 + threadIdx.y] = local_ar;
        x1px2[threadIdx.z * 4 + threadIdx.y] = local_al * local_ar;
    }
    __syncthreads();

    // Calculate v using extended EV and store results in x3
    double val = x1px2[threadIdx.y + threadIdx.z * 4] * extEV[threadIdx.y * 4 + lane];

    // Reduce final results within each warp
    val += __shfl_down_sync(0xFFFFFFFF, val, 4);

    if (threadIdx.y == 0) {
        x3[offset + lane] = val;
    }
}