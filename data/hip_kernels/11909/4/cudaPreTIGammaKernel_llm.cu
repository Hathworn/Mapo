#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

static unsigned int GRID_SIZE_N;
static unsigned int GRID_SIZE_4N;
static unsigned int MAX_STATE_VALUE;

__global__ static void cudaPreTIGammaKernel(double *tipVector, double *l, double *ump)
{
    __shared__ volatile double sump[64];
    const int tid = threadIdx.y * 4 + threadIdx.x;
    
    // Optimize memory access for coalescing
    double local_val = tipVector[4 * blockIdx.x + threadIdx.x] * l[tid];
    sump[tid] = local_val;
    __syncthreads();
    
    if (threadIdx.x <= 1)
    {
        sump[tid] += sump[tid + 2];
    }
    __syncthreads();
    
    if (threadIdx.x == 0)
    {
        sump[tid] += sump[tid + 1];
        // Directly store result to global memory
        ump[blockIdx.x * 16 + threadIdx.y] = sump[tid];
    }
}