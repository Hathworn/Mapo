#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

static unsigned int GRID_SIZE_N;
static unsigned int GRID_SIZE_4N;
static unsigned int MAX_STATE_VALUE;

__global__ static void cudaEvaluateRightGammaKernel(int *wptr, double *x1, double *x2, double *diagptable, double *output, const int limit) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Early return to avoid unnecessary computation
    if (i >= limit) {
        return;
    }

    output[i] = 0.0;

    int j;
    double term = 0.0;
    double tempTerm;
    
    // Use shared memory if available to improve data locality
    x1 += 16 * i;
    x2 += 16 * i;

    #pragma unroll
    for (j = 0; j < 4; j++) {
        // Vectorized operations can give better performance
        tempTerm  = x1[0] * x2[0] * diagptable[0];
        tempTerm += x1[1] * x2[1] * diagptable[1];
        tempTerm += x1[2] * x2[2] * diagptable[2];
        tempTerm += x1[3] * x2[3] * diagptable[3];
        
        term += tempTerm;
        
        x1 += 4;
        x2 += 4;
        diagptable += 4;
    }
    
    term = log(0.25 * fabs(term));
    output[i] += wptr[i] * term;
}