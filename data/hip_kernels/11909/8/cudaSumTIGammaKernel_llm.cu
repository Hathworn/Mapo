#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

static unsigned int GRID_SIZE_N;
static unsigned int GRID_SIZE_4N;
static unsigned int MAX_STATE_VALUE;

__global__ static void cudaSumTIGammaKernel(unsigned char *tipX1, double *x2, double *tipVector, double *sumtable, int limit) {
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= limit) return;

    const int i = n / 4, l = n % 4;
    double *left = &(tipVector[4 * tipX1[i]]);
    double *right = &(x2[16 * i + l * 4]);
    double *sum = &sumtable[i * 16 + l * 4];

    // Using local variables to minimize memory access latency
    double l0 = left[0], l1 = left[1], l2 = left[2], l3 = left[3];
    double r0 = right[0], r1 = right[1], r2 = right[2], r3 = right[3];

    // Unrolled loop for SIMD-like operations
    sum[0] = l0 * r0;
    sum[1] = l1 * r1;
    sum[2] = l2 * r2;
    sum[3] = l3 * r3;
}