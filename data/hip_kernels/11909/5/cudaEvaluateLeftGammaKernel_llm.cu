#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

static unsigned int GRID_SIZE_N;
static unsigned int GRID_SIZE_4N;
static unsigned int MAX_STATE_VALUE;

__global__ static void cudaEvaluateLeftGammaKernel(int *wptr, double *x2, double *tipVector, unsigned char *tipX1, double *diagptable, double *output, const int limit) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= limit) {
        output[i] = 0.0;
        return;
    }

    int j;
    double term = 0.0;
    double *tipVecPtr = tipVector + 4 * tipX1[i];
    double *x2Ptr = x2 + 16 * i;
    double *diagPtr = diagptable;
    
    #pragma unroll 4  // Unroll to improve loop efficiency
    for (j = 0; j < 4; j++) {
        term += tipVecPtr[0] * x2Ptr[0] * diagPtr[0];
        term += tipVecPtr[1] * x2Ptr[1] * diagPtr[1];
        term += tipVecPtr[2] * x2Ptr[2] * diagPtr[2];
        term += tipVecPtr[3] * x2Ptr[3] * diagPtr[3];
        
        x2Ptr += 4;
        diagPtr += 4;
    }

    term = log(0.25 * fabs(term));
    output[i] = wptr[i] * term;
}