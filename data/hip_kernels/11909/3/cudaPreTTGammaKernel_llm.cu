#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

static unsigned int GRID_SIZE_N;
static unsigned int GRID_SIZE_4N;
static unsigned int MAX_STATE_VALUE;

__global__ static void cudaPreTTGammaKernel(double *tipVector, double *l, double *r, double *umpX1, double *umpX2)
{
    __shared__ volatile double ump[64];
    const int tid = threadIdx.y * 4 + threadIdx.x;
    double tipValue = tipVector[4 * blockIdx.x + threadIdx.x];
    
    // Use conditional operator to eliminate branch divergence
    ump[tid] = tipValue * (blockIdx.y == 0 ? l[tid] : r[tid]);
    
    __syncthreads();
    
    // Use cooperative thread reduction for better performance
    if (threadIdx.x < 2) ump[tid] += ump[tid + 2];
    __syncthreads();
    
    if (threadIdx.x == 0) {
        ump[tid] += ump[tid + 1];
        if (blockIdx.y == 0) {
            umpX1[blockIdx.x * 16 + threadIdx.y] = ump[tid];
        } else {
            umpX2[blockIdx.x * 16 + threadIdx.y] = ump[tid];
        }
    }
}