#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelPowerTwo(const float *a, const float *b, const unsigned int length, float *a2, float *b2)
{
    // Calculate global thread index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index is within array bounds
    if (idx < length) {
        // Efficiently calculate squares
        a2[idx] = a[idx] * a[idx];
        b2[idx] = b[idx] * b[idx];
    }
}