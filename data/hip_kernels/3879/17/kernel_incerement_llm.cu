#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_increment(int *matrix, size_t pitch, int colCount, int rowCount)
{
    // Use shared memory for accessing matrix elements efficiently
    extern __shared__ int sharedMem[];

    uint xOffset = blockIdx.x * blockDim.x + threadIdx.x;
    uint yOffset = blockIdx.y * blockDim.y + threadIdx.y;

    // Load data into shared memory
    if (xOffset < colCount && yOffset < rowCount)
    {
        int *memoryRow = (int *)((char *)matrix + xOffset * pitch);
        sharedMem[threadIdx.y * blockDim.x + threadIdx.x] = memoryRow[yOffset];
    }
    __syncthreads();

    // Perform increment operation
    if (xOffset < colCount && yOffset < rowCount)
    {
        sharedMem[threadIdx.y * blockDim.x + threadIdx.x]++;
    }
    __syncthreads();

    // Write results back to global memory
    if (xOffset < colCount && yOffset < rowCount)
    {
        int *memoryRow = (int *)((char *)matrix + xOffset * pitch);
        memoryRow[yOffset] = sharedMem[threadIdx.y * blockDim.x + threadIdx.x];
    }
}