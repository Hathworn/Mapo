#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(const uint *__restrict__ a, const uint *__restrict__ b, const uint length, uint *c)
{
    // Load data into shared memory to improve memory efficiency
    extern __shared__ uint shared_a[];
    extern __shared__ uint shared_b[];

    uint tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint stride = blockDim.x * gridDim.x;

    // Each thread loads its part of the input arrays into shared memory
    if (tid < length) {
        shared_a[threadIdx.x] = a[tid];
        shared_b[threadIdx.x] = b[tid];
    }
    __syncthreads();  // Synchronize to ensure shared memory is properly loaded

    // Perform computation using shared memory
    while (tid < length) {
        c[tid] = shared_a[threadIdx.x] + shared_b[threadIdx.x];
        tid += stride;
    }
}