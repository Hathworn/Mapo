#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <functional>
#include <math.h>
#include <time.h>
#include <random>
#include <assert.h>
#include <>

typedef unsigned char byte;

static void HandleError(hipError_t err,
                        const char *file,
                        int line)
{
    if (err != hipSuccess)
    {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line);
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))
#define CUDA_CALL(err) (HandleError(err, __FILE__, __LINE__))

inline size_t get_number_of_parts(size_t whole, size_t divider)
{
    return ((whole + divider - 1) / divider);
}

#define CUDA_TIMED_BLOCK_START(fn_name)      \
    const char *___tmdFnName = fn_name;      \
    hipEvent_t startEvent, stopEvent;       \
    float elapsedTime;                       \
    CUDA_CALL(hipEventCreate(&startEvent)); \
    CUDA_CALL(hipEventCreate(&stopEvent));  \
    CUDA_CALL(hipEventRecord(startEvent, 0));

#define CUDA_TIMED_BLOCK_END                                              \
    CUDA_CALL(hipEventRecord(stopEvent, 0));                             \
    CUDA_CALL(hipEventSynchronize(stopEvent));                           \
    CUDA_CALL(hipEventElapsedTime(&elapsedTime, startEvent, stopEvent)); \
    printf("%s took: %f ms\n", ___tmdFnName, elapsedTime);                \
    CUDA_CALL(hipEventDestroy(startEvent));                              \
    CUDA_CALL(hipEventDestroy(stopEvent));

template <typename T>
bool all_not_eq(const std::vector<T> &data, const T &cmp)
{
    for (size_t i = 0; i < data.size(); i++)
    {
        if (data[i] == cmp)
            return false;
    }
    return true;
}

template <typename T>
void safe_cuda_free(T *ptr)
{
    if (ptr != nullptr)
    {
        hipFree(ptr);
    }
}
