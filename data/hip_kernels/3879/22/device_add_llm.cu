#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_add(int *a, int *b, int *c, int arrSize)
{
    int blockId = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global index
    if (blockId < arrSize)
    {
        c[blockId] = a[blockId] + b[blockId];  // Add elements
    }
}