#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(const float3 *__restrict__ dFinalForce, const unsigned int noRainDrops, float3 *__restrict__ dRainDrops)
{
    // Calculate global thread index
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by avoiding xSkip computation inside the loop
    uint gridSize = gridDim.x * blockDim.x;

    // Unroll loop to reduce loop overhead
    for (uint i = idx; i < noRainDrops; i += gridSize) {
        dRainDrops[i].x += dFinalForce->x;
        dRainDrops[i].y += dFinalForce->y;
        dRainDrops[i].z += dFinalForce->z;
    }
}