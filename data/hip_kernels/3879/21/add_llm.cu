#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c)
{
    // Compute only when threadIdx.x is 0 to avoid redundant writes
    if (threadIdx.x == 0) {
        *c = a + b;
    }
}