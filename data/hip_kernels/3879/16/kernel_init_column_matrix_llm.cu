#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_init_column_matrix(int *matrix, size_t pitch, size_t colCount, size_t rowCount)
{
    // Calculate unique thread index
    uint xOffset = blockIdx.x * blockDim.x + threadIdx.x;
    uint yOffset = blockIdx.y * blockDim.y + threadIdx.y;

    // Optimize by using stride for better memory coalescing
    uint strideX = gridDim.x * blockDim.x;
    uint strideY = gridDim.y * blockDim.y;

    // Loop with strided access
    for (uint x = xOffset; x < colCount; x += strideX) {
        for (uint y = yOffset; y < rowCount; y += strideY) {
            int *memoryRow = (int *)((char *)matrix + (x * pitch));
            memoryRow[y] = x * rowCount + y;
        }
    }
}