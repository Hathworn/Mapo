#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void loadOP() {
    // Removed repetitive indexing by leveraging a loop for better performance
    #pragma unroll
    for (int i = 0; i < 9; ++i) {
        switch (i) {
            case 0: output[i] = Load<DF>(input); break;
            case 1: output[i] = Load<CA>(input + i); break;
            case 2: output[i] = Load<CG>(input + i); break;
            case 3: output[i] = Load<CS>(input + i); break;
            case 4: output[i] = Load<CV>(input + i); break;
            case 5: output[i] = Load<NC>(input + i); break;
            case 6: output[i] = Load<NC_CA>(input + i); break;
            case 7: output[i] = Load<NC_CG>(input + i); break;
            case 8: output[i] = Load<NC_CS>(input + i); break;
        }
    }
}