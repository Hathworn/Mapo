#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prova3() {
    // Declaring shared memory to improve performance
    __shared__ int sharedData[32];

    // Assign a value to sharedData
    sharedData[threadIdx.x] = F_array[threadIdx.x];

    // Ensure all threads write to shared memory before reading
    __syncthreads();

    // Performing some operation on shared data
    if (threadIdx.x < 5) {
        Ouptput[threadIdx.x] = sharedData[threadIdx.x] + Adj_array[threadIdx.x];
    }
}
```
