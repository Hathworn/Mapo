#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void use_ptr3() {
    // Optimize by using threadIdx combined with blockIdx for parallel computation
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < 100) {
        const_ptr[idx] = idx;
    }
}