#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Avoid potential race condition by using atomic operation
    if (threadIdx.x == 0) {
        atomicExch(accuracy, *accuracy / N);
    }
}