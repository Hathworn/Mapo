#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AddAndRefreshConnectionKernel(int node1, int node2, int *activityFlag, int *connection, int *age, int maxCells)
{
    // Use single thread launch (threadId == 0) for the entire operation
    if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    {
        activityFlag[node1] = 1;
        activityFlag[node2] = 1;

        connection[node1 * maxCells + node2] = 1;
        age[node1 * maxCells + node2] = 0;
        connection[node2 * maxCells + node1] = 1;
        age[node2 * maxCells + node1] = 0;
    }
}