#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ChangeRecurrentWeightsKernel( float *recurrentWeights, float *recurrentWeightDeltas, float *outputWeights, float *outputDeltas, float *recurrentWeightRTRLDerivatives,  float trainingRate, float momentum )
{
    int weightId = blockDim.x * blockIdx.y * gridDim.x // rows preceding current row in grid
                 + blockDim.x * blockIdx.x             // blocks preceding current block
                 + threadIdx.x;

    if (weightId < D_HIDDEN_UNITS * D_HIDDEN_UNITS)
    {
        float gradient = 0.0f;
        
        // Use unrolling if possible; ensure memory coalescing
        for (int i = 0; i < D_OUTPUT_UNITS; i++)
        {
            float sum = 0.0f;
            for (int j = 0; j < D_HIDDEN_UNITS; j++)
            {
                sum += outputWeights[i * D_HIDDEN_UNITS + j] * recurrentWeightRTRLDerivatives[j * D_HIDDEN_UNITS * D_HIDDEN_UNITS + weightId];
            }
            gradient += outputDeltas[i] * sum;
        }

        // Efficient update pattern
        float weightDelta = trainingRate * gradient + momentum * recurrentWeightDeltas[weightId];
        recurrentWeightDeltas[weightId] = weightDelta;
        recurrentWeights[weightId] += weightDelta;
    }
}