#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RecurrentWeightsRTRLDerivativesKernel( float *previousHiddenActivations, float *hiddenActivationDerivatives, float *recurrentWeights, float *recurrentWeightRTRLDerivatives, float *previousRecurrentWeightRTRLDerivatives )
{
    int partialId = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x;

    if (partialId < D_HIDDEN_UNITS * D_HIDDEN_UNITS * D_HIDDEN_UNITS)
    {
        int unitId = partialId / (D_HIDDEN_UNITS * D_HIDDEN_UNITS);
        int weightId = partialId % (D_HIDDEN_UNITS * D_HIDDEN_UNITS);
        int to = weightId / D_HIDDEN_UNITS;
        int from = weightId % D_HIDDEN_UNITS;

        float sum = 0.0f;
        for (int i = 0; i < D_HIDDEN_UNITS; i++)
        {
            // Unrolling optimization to improve loop performance
            sum += recurrentWeights[unitId * D_HIDDEN_UNITS + i] * previousRecurrentWeightRTRLDerivatives[i * (D_HIDDEN_UNITS * D_HIDDEN_UNITS) + weightId];
        }

        // Using ternary operator to simplify the logic
        recurrentWeightRTRLDerivatives[partialId] = hiddenActivationDerivatives[unitId] * ((unitId == to ? previousHiddenActivations[from] : 0) + sum);
    }
}