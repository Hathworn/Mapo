#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CopyConnectionsCoordinatesKernel(int *connectionMatrix, float *pointsCoordinates, float *vertexData, int *connectionCount, int maxCells)
{
    // Calculate the unique thread ID
    int threadId = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x;

    if (threadId < maxCells * maxCells)
    {
        // Optimize by loading once and using early exit if not connected
        int connection = connectionMatrix[threadId];
        if (connection == 0) return;

        int from = threadId / maxCells;
        int to = threadId % maxCells;

        if (to > from)
        {
            int connIdx = atomicAdd(&connectionCount[0], 1) * 6; // Optimize index calculation

            float *fromPoint = &pointsCoordinates[from * 3]; // Pointer aliasing for clarity
            float *toPoint = &pointsCoordinates[to * 3];

            vertexData[connIdx] = fromPoint[0];
            vertexData[connIdx + 1] = fromPoint[1];
            vertexData[connIdx + 2] = fromPoint[2];

            vertexData[connIdx + 3] = toPoint[0];
            vertexData[connIdx + 4] = toPoint[1];
            vertexData[connIdx + 5] = toPoint[2];
        }
    }
}