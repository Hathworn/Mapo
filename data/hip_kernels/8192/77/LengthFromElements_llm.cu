#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LengthFromElements(float* element1, float* element2, float* output, int count)
{
    // Optimize threadId calculation to remove multiplication for y-dimension
    int threadId = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Use if statement only when necessary
    if(threadId < count)
    {
        float val1 = element1[threadId];
        float val2 = element2[threadId];
        // Calculate without redundant multiplications
        output[threadId] = sqrtf(val1 * val1 + val2 * val2);
    }
}