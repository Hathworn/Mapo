#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AvgCentroidCoordinatesKernel(float *centroidCoordinates, float *pointsWeight, int inputSize, int centroids) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread ID calculation

    if (threadId < centroids * 2) {
        float weight = pointsWeight[threadId / 2]; // Load weight only once
        if (weight == 0.00f) {
            centroidCoordinates[threadId] = 0.00f;
        } else {
            centroidCoordinates[threadId] /= weight; // Division with preloaded weight
        }
    }
}