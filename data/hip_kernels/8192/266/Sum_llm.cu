#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Sum(float *A, float *B, float *C, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;   // Simplify thread index calculation
    int stride = blockDim.x * gridDim.x;               // Calculate stride for larger problems
    for (int id = idx; id < size; id += stride) {      // Loop for larger data sizes
        C[id] = A[id] + B[id];                         // Perform element-wise addition
    }
}