#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UpdateCC_XY(float *CCXY, int id_CC, float *XY_tofill, int dim_XY) {
    // Calculate global thread index more efficiently
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    if (id < dim_XY) {
        // Directly assign value without checking again
        CCXY[id_CC * dim_XY + id] = XY_tofill[id];
    }
}