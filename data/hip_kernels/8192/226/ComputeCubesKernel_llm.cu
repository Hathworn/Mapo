#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeCubesKernel(float *pointsCoordinates, float *vertexData, int quadOffset, float cubeSide, int *activityFlag, int textureWidth, int maxCells) {
    int threadId = blockDim.x * blockIdx.y * gridDim.x
                 + blockDim.x * blockIdx.x
                 + threadIdx.x;

    if (threadId < maxCells) {
        float x = pointsCoordinates[threadId * 3];
        float y = pointsCoordinates[threadId * 3 + 1];
        float z = pointsCoordinates[threadId * 3 + 2];

        float halfSide = (activityFlag[threadId] == 0) ? 0.00f : 0.50f * cubeSide;

        const int vertexStride = 72;
        const int textureStride = 48;
        int vertexBase = quadOffset + threadId * vertexStride;
        int textureBase = quadOffset + maxCells * 4 * 6 * 3 + threadId * textureStride;

        float textureAbsLength = (float)(maxCells * textureWidth);
        float t1 = (float)((threadId) * textureWidth) / textureAbsLength;
        float t2 = (float)((threadId + 1) * textureWidth) / textureAbsLength;

        // Pre-compute vertex offsets for reusability - DRY principle
        int vBase[6] = {0, 12, 24, 36, 48, 60};
        float verts[8][3] = {
            {x - halfSide, y - halfSide, z - halfSide}, {x + halfSide, y - halfSide, z - halfSide}, 
            {x - halfSide, y + halfSide, z - halfSide}, {x + halfSide, y + halfSide, z - halfSide},
            {x - halfSide, y - halfSide, z + halfSide}, {x + halfSide, y - halfSide, z + halfSide},
            {x - halfSide, y + halfSide, z + halfSide}, {x + halfSide, y + halfSide, z + halfSide}
        };

        // BOTTOM SIDE
        vertexData[vertexBase + vBase[0] + 0] = verts[4][0];
        vertexData[vertexBase + vBase[0] + 1] = verts[4][1];
        vertexData[vertexBase + vBase[0] + 2] = verts[4][2];
        vertexData[vertexBase + vBase[0] + 3] = verts[0][0];
        vertexData[vertexBase + vBase[0] + 4] = verts[0][1];
        vertexData[vertexBase + vBase[0] + 5] = verts[0][2];
        vertexData[vertexBase + vBase[0] + 6] = verts[1][0];
        vertexData[vertexBase + vBase[0] + 7] = verts[1][1];
        vertexData[vertexBase + vBase[0] + 8] = verts[1][2];
        vertexData[vertexBase + vBase[0] + 9] = verts[5][0];
        vertexData[vertexBase + vBase[0] + 10] = verts[5][1];
        vertexData[vertexBase + vBase[0] + 11] = verts[5][2];

        // FRONT SIDE
        vertexData[vertexBase + vBase[1] + 0] = verts[6][0];
        vertexData[vertexBase + vBase[1] + 1] = verts[6][1];
        vertexData[vertexBase + vBase[1] + 2] = verts[6][2];
        vertexData[vertexBase + vBase[1] + 3] = verts[4][0];
        vertexData[vertexBase + vBase[1] + 4] = verts[4][1];
        vertexData[vertexBase + vBase[1] + 5] = verts[4][2];
        vertexData[vertexBase + vBase[1] + 6] = verts[5][0];
        vertexData[vertexBase + vBase[1] + 7] = verts[5][1];
        vertexData[vertexBase + vBase[1] + 8] = verts[5][2];
        vertexData[vertexBase + vBase[1] + 9] = verts[7][0];
        vertexData[vertexBase + vBase[1] + 10] = verts[7][1];
        vertexData[vertexBase + vBase[1] + 11] = verts[7][2];

        // LEFT SIDE
        vertexData[vertexBase + vBase[2] + 0] = verts[6][0];
        vertexData[vertexBase + vBase[2] + 1] = verts[6][1];
        vertexData[vertexBase + vBase[2] + 2] = verts[6][2];
        vertexData[vertexBase + vBase[2] + 3] = verts[2][0];
        vertexData[vertexBase + vBase[2] + 4] = verts[2][1];
        vertexData[vertexBase + vBase[2] + 5] = verts[2][2];
        vertexData[vertexBase + vBase[2] + 6] = verts[4][0];
        vertexData[vertexBase + vBase[2] + 7] = verts[4][1];
        vertexData[vertexBase + vBase[2] + 8] = verts[4][2];
        vertexData[vertexBase + vBase[2] + 9] = verts[0][0];
        vertexData[vertexBase + vBase[2] + 10] = verts[0][1];
        vertexData[vertexBase + vBase[2] + 11] = verts[0][2];

        // BACK SIDE
        vertexData[vertexBase + vBase[3] + 0] = verts[3][0];
        vertexData[vertexBase + vBase[3] + 1] = verts[3][1];
        vertexData[vertexBase + vBase[3] + 2] = verts[3][2];
        vertexData[vertexBase + vBase[3] + 3] = verts[2][0];
        vertexData[vertexBase + vBase[3] + 4] = verts[2][1];
        vertexData[vertexBase + vBase[3] + 5] = verts[2][2];
        vertexData[vertexBase + vBase[3] + 6] = verts[1][0];
        vertexData[vertexBase + vBase[3] + 7] = verts[1][1];
        vertexData[vertexBase + vBase[3] + 8] = verts[1][2];
        vertexData[vertexBase + vBase[3] + 9] = verts[0][0];
        vertexData[vertexBase + vBase[3] + 10] = verts[0][1];
        vertexData[vertexBase + vBase[3] + 11] = verts[0][2];

        // RIGHT SIDE
        vertexData[vertexBase + vBase[4] + 0] = verts[7][0];
        vertexData[vertexBase + vBase[4] + 1] = verts[7][1];
        vertexData[vertexBase + vBase[4] + 2] = verts[7][2];
        vertexData[vertexBase + vBase[4] + 3] = verts[5][0];
        vertexData[vertexBase + vBase[4] + 4] = verts[5][1];
        vertexData[vertexBase + vBase[4] + 5] = verts[5][2];
        vertexData[vertexBase + vBase[4] + 6] = verts[3][0];
        vertexData[vertexBase + vBase[4] + 7] = verts[3][1];
        vertexData[vertexBase + vBase[4] + 8] = verts[3][2];
        vertexData[vertexBase + vBase[4] + 9] = verts[1][0];
        vertexData[vertexBase + vBase[4] + 10] = verts[1][1];
        vertexData[vertexBase + vBase[4] + 11] = verts[1][2];

        // UPPER SIDE
        vertexData[vertexBase + vBase[5] + 0] = verts[6][0];
        vertexData[vertexBase + vBase[5] + 1] = verts[6][1];
        vertexData[vertexBase + vBase[5] + 2] = verts[6][2];
        vertexData[vertexBase + vBase[5] + 3] = verts[2][0];
        vertexData[vertexBase + vBase[5] + 4] = verts[2][1];
        vertexData[vertexBase + vBase[5] + 5] = verts[2][2];
        vertexData[vertexBase + vBase[5] + 6] = verts[7][0];
        vertexData[vertexBase + vBase[5] + 7] = verts[7][1];
        vertexData[vertexBase + vBase[5] + 8] = verts[7][2];
        vertexData[vertexBase + vBase[5] + 9] = verts[3][0];
        vertexData[vertexBase + vBase[5] + 10] = verts[3][1];
        vertexData[vertexBase + vBase[5] + 11] = verts[3][2];

        // Update texture coordinates
        float texCoords[2][4] = {{t1, 0.00f, t1, 1.00f}, {t2, 1.00f, t2, 0.00f}};
        for (int i = 0; i < 6; i++) {
            vertexData[textureBase + i * 8 + 0] = texCoords[0][0];
            vertexData[textureBase + i * 8 + 1] = texCoords[0][1];
            vertexData[textureBase + i * 8 + 2] = texCoords[0][2];
            vertexData[textureBase + i * 8 + 3] = texCoords[0][3];
            vertexData[textureBase + i * 8 + 4] = texCoords[1][0];
            vertexData[textureBase + i * 8 + 5] = texCoords[1][1];
            vertexData[textureBase + i * 8 + 6] = texCoords[1][2];
            vertexData[textureBase + i * 8 + 7] = texCoords[1][3];
        }
    }
}