#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SumCentroids(float* delta, float* sumDelta, int numOfCentroids, int numOfElements)
{
    // Calculate unique thread id
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure id is within valid range
    if (id < numOfCentroids * NUM_SUMS)
    {
        float sum = 0;
        int offset = id * numOfElements;

        // Optimize loop by unrolling for better memory access patterns (assumes numOfElements is a multiple of 4)
        for (int i = 0; i < numOfElements; i += 4)
        {
            sum += delta[offset + i];
            sum += delta[offset + i + 1];
            sum += delta[offset + i + 2];
            sum += delta[offset + i + 3];
        }
        sumDelta[id] = sum;
    }
}