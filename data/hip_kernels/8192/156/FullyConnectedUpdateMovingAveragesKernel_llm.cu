#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FullyConnectedUpdateMovingAveragesKernel( float *weightsGradPtr, float *biasGradPtr, float *weightsGradCurvePtr, float *biasGradCurvePtr, float *avgWeightGradPtr, float *avgBiasGradPtr, float *avgWeightGradVarPtr, float *avgBiasGradVarPtr, float *avgWeightGradCurvePtr, float *avgBiasGradCurvePtr, float *avgWeightGradCurveVarPtr, float *avgBiasGradCurveVarPtr, float *weightMemorySizePtr, float *biasMemorySizePtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize )
{
    // Calculate j based on thread location
    int j = threadIdx.x + blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x);

    if (j < thisLayerSize && !dropoutMaskPtr[j])
    {
        // Compute index for weight operations
        int index = j;
        
        for (int i = 0; i < prevLayerSize; i++, index += thisLayerSize)
        {
            float weightMemoryInv = 1.0f / weightMemorySizePtr[index];
            float weightAvgFactor = 1.0f - weightMemoryInv;

            // Update moving averages for weights
            avgWeightGradPtr[index] = weightAvgFactor * avgWeightGradPtr[index] + weightMemoryInv * weightsGradPtr[index];
            avgWeightGradVarPtr[index] = weightAvgFactor * avgWeightGradVarPtr[index] + weightMemoryInv * weightsGradPtr[index] * weightsGradPtr[index];
            avgWeightGradCurvePtr[index] = weightAvgFactor * avgWeightGradCurvePtr[index] + weightMemoryInv * weightsGradCurvePtr[index];
            avgWeightGradCurveVarPtr[index] = weightAvgFactor * avgWeightGradCurveVarPtr[index] + weightMemoryInv * weightsGradCurvePtr[index] * weightsGradCurvePtr[index];
        }

        float biasMemoryInv = 1.0f / biasMemorySizePtr[j];
        float biasAvgFactor = 1.0f - biasMemoryInv;

        // Update moving averages for biases
        avgBiasGradPtr[j] = biasAvgFactor * avgBiasGradPtr[j] + biasMemoryInv * biasGradPtr[j];
        avgBiasGradVarPtr[j] = biasAvgFactor * avgBiasGradVarPtr[j] + biasMemoryInv * biasGradPtr[j] * biasGradPtr[j];
        avgBiasGradCurvePtr[j] = biasAvgFactor * avgBiasGradCurvePtr[j] + biasMemoryInv * biasGradCurvePtr[j];
        avgBiasGradCurveVarPtr[j] = biasAvgFactor * avgBiasGradCurveVarPtr[j] + biasMemoryInv * biasGradCurvePtr[j] * biasGradCurvePtr[j];
    }
}