#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeBiasedDistanceKernel(float *distance, float *biasedDistance, float *biasTerm, int maxCells)
{
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x; // Calculate global thread ID

    if (threadId < maxCells)
    {
        biasedDistance[threadId] = distance[threadId] + biasTerm[threadId]; // Compute biased distance
    }
}