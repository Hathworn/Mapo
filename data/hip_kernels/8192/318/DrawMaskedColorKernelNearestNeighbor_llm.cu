#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DrawMaskedColorKernelNearestNeighbor(float *target, int targetWidth, int targetHeight, int inputX, int inputY, float *texture, int textureWidth, int textureHeight, int objectWidth, int objectHeight, float r, float g, float b)
{
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    int targetPixels = targetWidth * targetHeight;
    int texturePixels = textureWidth * textureHeight;
    int objectPixels = objectWidth * objectHeight;

    // Calculate object RGB and pixel index
    int idObjectRgb = id / objectPixels;
    int idObjectPixel = id % objectPixels;
    int idObjectY = idObjectPixel / objectWidth;
    int idObjectX = idObjectPixel % objectWidth;

    if (idObjectRgb < 3) 
    {
        // Calculate target RGB component
        int targetRgb = (idObjectRgb == 0) ? 2 : ((idObjectRgb == 2) ? 0 : 1);

        // Check boundary conditions
        if (idObjectX + inputX < targetWidth && idObjectX + inputX >= 0 &&
            idObjectY + inputY < targetHeight && idObjectY + inputY >= 0)
        {
            // Compute nearest neighbor texture coordinates
            int textureX = textureWidth * idObjectX / objectWidth;
            int textureY = textureHeight * idObjectY / objectHeight;
            int textureId = textureY * textureWidth + textureX;
            int aIndex = textureId + 3 * texturePixels;

            float a = texture[aIndex];

            if (a > 0) 
            {
                int tIndex = targetPixels * targetRgb + targetWidth * (idObjectY + inputY) + (idObjectX + inputX);

                // Apply color blending
                target[tIndex] = target[tIndex] * (1.0f - a);

                switch (idObjectRgb)
                {
                case 0:
                    target[tIndex] += a * r;
                    break;
                case 1:
                    target[tIndex] += a * g;
                    break;
                case 2:
                    target[tIndex] += a * b;
                    break;
                }
            }
        }
    }
}
```
