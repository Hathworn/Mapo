#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DrawRgbBackgroundKernel(float *target, int inputWidth, int inputHeight, float r, float g, float b)
{
    int column = threadIdx.x + blockDim.x * blockIdx.z;
    if (column >= inputWidth) return; // Early exit if column out of bounds

    int row = blockIdx.x; // Use blockIdx.x for row
    int colorChannel = blockIdx.y; // Use blockIdx.y for color channel
    int imagePixels = inputWidth * inputHeight;
    int id = column + row * inputWidth + colorChannel * imagePixels; // Calculate id with rearranged operations

    if (id < 3 * imagePixels)
    {
        float color = (colorChannel == 0) ? r : (colorChannel == 1) ? g : b; // Use conditional operator for color
        target[id] = color;
    }
}