#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CopyRectangleCheckBoundsKernel(
    float *src, int srcOffset, int srcWidth, int srcHeight, 
    int srcRectX, int srcRectY, int rectWidth, int rectHeight, 
    float *dest, int destOffset, int destWidth, 
    int destRectX, int destRectY, float defaultValue) 
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x; 

    // Calculate grid-stride loop bounds
    int size = rectWidth * rectHeight; 

    // Loop over all elements to support grids larger than input size
    for (int index = id; index < size; index += blockDim.x * gridDim.x) {
        int localX = index % rectWidth;
        int localY = index / rectWidth;
        
        int srcPixelX = srcRectX + localX;
        int srcPixelY = srcRectY + localY;

        int destPixelX = destRectX + localX;
        int destPixelY = destRectY + localY;

        // Check if source pixel is within bounds before copying
        if (srcPixelX >= 0 && srcPixelX < srcWidth && srcPixelY >= 0 && srcPixelY < srcHeight) {
            (dest + destOffset)[destPixelX + destPixelY * destWidth] =
                (src + srcOffset)[srcPixelX + srcPixelY * srcWidth];
        } else {
            (dest + destOffset)[destPixelX + destPixelY * destWidth] = defaultValue;
        }
    }
}