#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CumulatePositionOfNewObjects(float* mask, float* maskNewIds, float* maskOut, int mask_size, int mask_cols, float* centers, int centers_size, int centers_columns) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate linear index for 1D grid

    if (idx < mask_size) {
        int icol = idx % mask_cols;  // Calculate column index
        int irow = idx / mask_cols;  // Calculate row index

        int i_mask = mask[idx];
        int i_obj = maskNewIds[i_mask];
        maskOut[idx] = i_obj;

        if (i_obj * centers_columns + 2 < centers_size) {
            // Use shared memory to accumulate center updates
            float center_updates[3] = {0.0f, 0.0f, 0.0f};
            center_updates[0] = (float)icol;
            center_updates[1] = (float)irow;
            center_updates[2] = 1.0f;

            atomicAdd(centers + 0 + i_obj * centers_columns, center_updates[0]);
            atomicAdd(centers + 1 + i_obj * centers_columns, center_updates[1]);
            atomicAdd(centers + 2 + i_obj * centers_columns, center_updates[2]);
        }
    }
}