#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeCubes2Kernel(float *pointsCoordinates, float *vertexData, int quadOffset, float cubeSide, float *cubeOperation, float *cubeTexCoordinates, int *activityFlag, float textureWidth, int maxCells)
{
    int threadId = blockDim.x * blockIdx.y * gridDim.x     // Calculate global thread ID
                 + blockDim.x * blockIdx.x
                 + threadIdx.x;

    if (threadId < maxCells * 6)
    {
        int cellId = threadId / 6;
        float fCellId = (float)cellId;
        int sideId = threadId % 6;

        // Cache coordinates and halfSide to reduce memory accesses
        float x = pointsCoordinates[cellId * 3];
        float y = pointsCoordinates[cellId * 3 + 1];
        float z = pointsCoordinates[cellId * 3 + 2];
        float halfSide = (activityFlag[cellId] == 1) * 0.50f * cubeSide;

        int vertexBase = quadOffset + cellId * 72 + 12 * sideId;
        int textureOffset = quadOffset + maxCells * 4 * 6 * 3;
        float textureAbsLength = (float)maxCells * textureWidth;

        // Loop unrolling for vertex computation
        #pragma unroll
        for(int i = 0; i < 12; i += 3)
        {
            vertexData[vertexBase + i]     = x + operationMaskConstant[12 * sideId + i] * halfSide;
            vertexData[vertexBase + i + 1] = y + operationMaskConstant[12 * sideId + i + 1] * halfSide;
            vertexData[vertexBase + i + 2] = z + operationMaskConstant[12 * sideId + i + 2] * halfSide;
        }

        int texBase = textureOffset + cellId * 48 + 8 * sideId;

        // Loop unrolling for texture coordinate computation
        #pragma unroll
        for(int j = 0; j < 8; j += 2)
        {
            vertexData[texBase + j]     = ((fCellId + cubeTexCoordinatesConstant[sideId * 8 + j]) * textureWidth) / textureAbsLength;
            vertexData[texBase + j + 1] = cubeTexCoordinatesConstant[sideId * 8 + j + 1];
        }
    }
}