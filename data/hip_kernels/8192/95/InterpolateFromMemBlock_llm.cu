#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InterpolateFromMemBlock(float* __restrict__ input1, float* __restrict__ input2, float* __restrict__ output, float* __restrict__ weightMemBlock, int inputSize)
{
    // Calculate unique thread ID across grid
    int threadId = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x; 

    // Ensure threadId is within bounds
    if (threadId < inputSize)
    {
        float weight = weightMemBlock[0]; // Cache shared weight to register
        float in1 = input1[threadId];
        float in2 = input2[threadId];

        // Use ternary operator for conditional interpolation
        output[threadId] = (weight <= 0) ? in1 : (weight >= 1) ? in2 : (1 - weight) * in1 + weight * in2;
    }
}