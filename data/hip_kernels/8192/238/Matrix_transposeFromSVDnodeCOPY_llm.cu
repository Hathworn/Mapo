#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_transposeFromSVDnodeCOPY(const float* A, int Acount, int Acols, float* out0)
{
    // Calculate global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int Arows = Acount / Acols;

    // Loop to allow processing more data elements per thread
    for (int id = idx; id < Acount; id += stride)
    {
        int x = id / Arows; // Column index
        int y = id % Arows; // Row index

        out0[x * Arows + y] = A[y * Acols + x];
    }
}