#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PIDControllerKernel(float* input, float* goal, float* output, float* previousError, float* integral) 
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified calculation of global thread ID.

    if (id < D_COUNT) 
    {
        float error = input[id] - goal[id];
        integral[id] = D_INTEGRAL_DECAY * integral[id] + error;  // Keep integral update concise.
        float derivative = error - previousError[id];

        previousError[id] = error;  // Update previousError.

        float out = D_OFFSET + D_PROPORTIONAL_GAIN * error 
                    + D_INTEGRAL_GAIN * integral[id] 
                    + D_DERIVATIVE_GAIN * derivative;  // Compute PID output.
        
        // Clamp the output to be within min and max limits.
        out = fminf(fmaxf(out, D_MIN_OUTPUT), D_MAX_OUTPUT);

        output[id] = out;
    }
}