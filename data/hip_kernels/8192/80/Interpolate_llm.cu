#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized Interpolate kernel function
__global__ void Interpolate(float* input1, float* input2, float* output, float weight, int inputSize)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Calculate thread ID

    // Ensure thread is within bounds
    if (threadId < inputSize)
    {
        // Precompute weight coefficients
        float weight1 = 1.0f - weight;
        float weight2 = weight;

        if (weight <= 0)
        {
            output[threadId] = input1[threadId];
        }
        else if (weight >= 1)
        {
            output[threadId] = input2[threadId];
        }
        else
        {
            // Perform interpolation with precomputed weights
            output[threadId] = weight1 * input1[threadId] + weight2 * input2[threadId];
        }
    }
}