#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GaussianMinMaxField(float* input, int inputCount, float* mins, float* maxes)
{
    // Calculate global index in a more straightforward way
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within input bounds
    if (i < inputCount)
    {
        float inputValue = input[i]; // Load input once to reduce global memory access
        atomicMin(&mins[i], inputValue); // Use atomic operations to safely handle concurrent writes
        atomicMax(&maxes[i], inputValue);
    }
}