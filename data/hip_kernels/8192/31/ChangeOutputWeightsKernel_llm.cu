#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ChangeOutputWeightsKernel( float *outputWeights, float *outputWeightDeltas, float *outputDeltas, float *hiddenActivations,  float trainingRate, float momentum )
{
    int weightId = blockIdx.x * blockDim.x + threadIdx.x; // Simplified calculation of weightId

    // Unroll loop if enough iterations are expected for optimization
    int totalWeights = D_OUTPUT_UNITS * D_HIDDEN_UNITS;
    if (weightId < totalWeights)
    {
        int to = weightId / D_HIDDEN_UNITS;
        int from = weightId % D_HIDDEN_UNITS;

        float gradient = outputDeltas[to] * hiddenActivations[from];
        float weightDelta = trainingRate * gradient + momentum * outputWeightDeltas[weightId];
        outputWeightDeltas[weightId] = weightDelta;
        outputWeights[weightId] += weightDelta;
    }
}