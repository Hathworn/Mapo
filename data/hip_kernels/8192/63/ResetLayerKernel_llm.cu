#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using 2D grid and block dimensions
__global__ void ResetLayerKernel(float *layer, float value, int count) 
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x 
                 + (blockIdx.y * gridDim.x * blockDim.x); // Simplified thread index calculation

    if (threadId < count) 
    {
        layer[threadId] = value; // Set the layer value
    }
}