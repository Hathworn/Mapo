#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FillAdjacencyMatrix(float* adj_mat, float* maskBuffer, int size, int cols, int rows, int Nsegs) {
    int idx = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    if (idx >= size) return; // Early exit if idx is out of bounds

    int icol = idx % cols;
    int irow = idx / cols;
    
    if (icol >= cols - 2 || irow >= rows - 2 || irow <= 1 || icol <= 1) return; // Boundary check

    int seg_id1 = maskBuffer[idx];
    if (seg_id1 != maskBuffer[idx + 1]) {
        int seg_id2 = maskBuffer[idx + 1];
        adj_mat[seg_id2 + seg_id1 * Nsegs] = 1; // Record adjacency
        adj_mat[seg_id1 + seg_id2 * Nsegs] = 1; // Ensure symmetry
    } else if (seg_id1 != maskBuffer[idx - cols]) {
        int seg_id2 = maskBuffer[idx - cols];
        adj_mat[seg_id2 + seg_id1 * Nsegs] = 1; // Record adjacency
        adj_mat[seg_id1 + seg_id2 * Nsegs] = 1; // Ensure symmetry
    }
}