#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void OFConvertXY2AngleSize (float* of, int id, int imageSize, float& of_size, float& of_angle) {
    float2 OF_value;
    OF_value.x = of[id];
    OF_value.y = of[id + imageSize];
   
    float sum_squares = (OF_value.x + OF_value.y) * (OF_value.x + OF_value.y);  // Reduce duplicate computation
    of_size = sqrtf(sum_squares);  // Use sqrtf for performance
    of_angle = atan2f(OF_value.x, OF_value.y);  // Use atan2f for performance
}

__global__ void OFConvert2AngleSize (float* of, int imageSize) {
    int id = blockIdx.x * blockDim.x + threadIdx.x  // Simplified indexing calculation
             + blockIdx.y * blockDim.x * gridDim.x;

    if (id < imageSize) {
        float OF_size;
        float OF_angle;
        
        OFConvertXY2AngleSize(of, id, imageSize, OF_size, OF_angle);

        of[id] = OF_angle;
        of[id + imageSize] = OF_size;
    }
}