#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void Matrix_getCol_FloatId_optimized(const float * A, int Acols, float * out0, int out0count, float col_id) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < out0count) {
        // Access target column directly more efficiently
        out0[id] = A[id * Acols + static_cast<int>(col_id)];
    }
}