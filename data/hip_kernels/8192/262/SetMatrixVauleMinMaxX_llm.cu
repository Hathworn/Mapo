#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SetMatrixVauleMinMaxX(float* matrix, int cols, int size, int id_min, int id_max, float value)
{
    // Calculate unique thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    // Process valid elements more efficiently
    if (id < size) {
        int id_column = id % cols;
        if (id_column >= id_min && id_column <= id_max) {
            matrix[id] = value;
        }
    }
}