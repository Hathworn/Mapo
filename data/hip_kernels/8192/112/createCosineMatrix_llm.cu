#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void createCosineMatrix(float* matrix, int xsize) {
    int threadGlobalID = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
        + threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;

    // Optimization: Removed redundant check inside loop by separating the logic
    if (threadGlobalID < xsize) {
        float cosFactor = (threadGlobalID == 0) ? 1 / sqrt((float)xsize) : (sqrt((float)2 / xsize));

        for (int i = 0; i < xsize; i++) {
            matrix[threadGlobalID + i * xsize] = (threadGlobalID == 0) 
                ? cosFactor 
                : cosFactor * cos((PI * (2 * i + 1) * threadGlobalID) / (2 * xsize));
        }
    }
}