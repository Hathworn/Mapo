#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SetMatrixValueMinMaxY(float* matrix, int cols, int size, int id_min, int id_max, float value)
{
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;  // Calculate global thread ID
    if (id < size)
    {
        int id_row = id / cols;  // Compute the row index
        if (id_row >= id_min && id_row <= id_max)
        {
            matrix[id] = value;  // Set value only if within the specified row range
        }
    }
}