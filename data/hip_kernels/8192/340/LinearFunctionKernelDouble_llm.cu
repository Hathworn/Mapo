#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LinearFunctionKernelDouble(double a1, double a0, double* input, double* output, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified global thread ID calculation

    if (id < size)
    {
        output[id] = a1 * input[id] + a0; // Direct computation without unnecessary variable
    }
}