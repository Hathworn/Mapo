#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CopyVectorKernel(float *from, int fromOffset, float *to, int toOffset, int vectorSize)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;  // Flattening 2D grid into 1D index

    if (threadId < vectorSize)
    {
        to[threadId + toOffset] = from[threadId + fromOffset];
    }
}