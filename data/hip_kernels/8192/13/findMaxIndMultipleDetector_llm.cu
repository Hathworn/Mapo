#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findMaxIndMultipleDetector(float *input, int* maxInd, int size)
{
    __shared__ float sharedMaxValue;
    __shared__ int sharedMaxIndex;

    if(threadIdx.x == 0) {
        sharedMaxValue = input[0];
        sharedMaxIndex = 0;
    }
    __syncthreads();

    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        if (input[i] > sharedMaxValue) {
            atomicExch(&sharedMaxValue, input[i]);
            atomicExch(&sharedMaxIndex, i);
        }
    }
    __syncthreads();

    if(threadIdx.x == 0) {
        int count = 0;
        for (int i = 0; i < size; i++) {
            if (input[i] == sharedMaxValue) {
                count++;
            }
        }
        if(count > 1)
            maxInd[0] = -1;
        else
            maxInd[0] = sharedMaxIndex;
    }
}