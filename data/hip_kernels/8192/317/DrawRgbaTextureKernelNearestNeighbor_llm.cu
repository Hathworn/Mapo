#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DrawRgbaTextureKernelNearestNeighbor(float *target, int targetWidth, int targetHeight, int inputX, int inputY, float *texture, int textureWidth, int textureHeight, int objectWidth, int objectHeight)
{
    int id = blockDim.x * blockIdx.y * gridDim.x
           + blockDim.x * blockIdx.x
           + threadIdx.x;

    int idObjectRgb = id / (objectWidth * objectHeight);
    int idObjectPixel = id % (objectWidth * objectHeight); // Optimized modulo operation
    int idObjectY = idObjectPixel / objectWidth;
    int idObjectX = idObjectPixel % objectWidth; // Optimized modulo operation

    if (idObjectRgb < 3) // 3 channels to write to (R, G, B)
    {
        int targetRgb = (idObjectRgb == 0) ? 2 : (idObjectRgb == 2) ? 0 : 1;

        // Check if the projected object pixel lies within target bounds
        if (idObjectX + inputX < targetWidth && idObjectX + inputX >= 0 &&
            idObjectY + inputY < targetHeight && idObjectY + inputY >= 0)
        {
            // Compute nearest neighbor coordinates
            int textureX = textureWidth * idObjectX / objectWidth;
            int textureY = textureHeight * idObjectY / objectHeight;
            int textureId = textureY * textureWidth + textureX;

            float textureValue = texture[textureId + idObjectRgb * textureWidth * textureHeight];

            int tIndex = targetWidth * targetHeight * targetRgb + targetWidth * (idObjectY + inputY) + (idObjectX + inputX);
            float a = texture[textureId + 3 * textureWidth * textureHeight];
            target[tIndex] = target[tIndex] * (1.0f - a) + a * textureValue;
        }
    }
}