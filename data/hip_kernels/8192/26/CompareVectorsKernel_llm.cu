#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CompareVectorsKernel(float* inputOne, float* inputTwo, float* output)
{
    // Calculate 1D global index
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use if condition for checking bounds
    if (id < gridDim.x * blockDim.x) {
        // Compare and write to output
        if (inputOne[id] != inputTwo[id]) {
            atomicExch(output, 1); // Use atomic operation to prevent race conditions
        }
    }
}