#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeHistogramKernel(float *globalMemData, int *globalHist) {
    // Calculate unique global thread ID
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    int localThreadId = threadIdx.x;
    
    // Declare shared memory for partial histogram
    extern __shared__ int partialHist[];

    // Initialize partial histogram in shared memory to zero
    if (localThreadId < D_BINS) {
        partialHist[localThreadId] = 0;
    }
    __syncthreads();

    // Ensure global thread id is within bounds
    if (globalThreadId < D_MEMORY_BLOCK_SIZE) {
        // Load data value from global memory
        float myLocalDataValue = globalMemData[globalThreadId];
        int binIdToWrite = 0;

        // Check if the local value is within limits
        if (myLocalDataValue >= D_MIN_VALUE && myLocalDataValue <= D_MAX_VALUE) {
            float biasedValue = myLocalDataValue - D_MIN_VALUE;
            binIdToWrite = (int)floorf(biasedValue / D_BIN_VALUE_WIDTH) + 1;
            if (myLocalDataValue == D_MAX_VALUE) {
                binIdToWrite = D_BINS - 2;
            }
        } else if (myLocalDataValue > D_MAX_VALUE) {
            binIdToWrite = D_BINS - 1;
        }
        
        // Update partial histogram in shared memory
        atomicAdd(&(partialHist[binIdToWrite]), 1);
    }
    __syncthreads();

    // Update global histogram with values from partial histogram
    if (localThreadId < D_BINS) {
        atomicAdd(&(globalHist[localThreadId]), partialHist[localThreadId]);
    }
}