#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize grid and block indexing calculation
__global__ void AdaptWinningFractionKernel(int s1, float *winningFraction, int *winningCount, float bParam, int maxCells)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread ID calculation

    if(threadId < maxCells)
    {
        float wf = winningFraction[threadId];
        float increment = bParam * ((float)(threadId == s1) - wf); // Cache repetitive calculation
        winningFraction[threadId] += increment; // Use compound assignment to modify in place
        winningCount[threadId] += (threadId == s1); // Simplified increment logic
    }
}