#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MaskByNaN(float* inputImage, float* mask, float* outputImage, int count) {
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    if (id < count) {
        outputImage[id] = (mask[id] == 0.0f) ? NAN : inputImage[id];  // Use ternary operator for simplicity
    }
}