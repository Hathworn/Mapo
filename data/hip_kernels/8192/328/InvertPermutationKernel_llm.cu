#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void InvertPermutationKernel(float* input, float* output, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified ID calculation with one-dimensional grid

    if (id < size) // Simplified boundary condition
    {
        int temp = __float2int_rn(input[id]);

        if (input == output) // Check if input is output
            __syncthreads();

        output[temp] = id;
    }
}