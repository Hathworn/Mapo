#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UniformNormalDistribution(float *from, float *to, int size)
{
    // Calculate global thread ID
    int id = threadIdx.x + blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x);

    // Check bounds and compute
    if (id < size) 
    {
        // Precompute constant outside loop (in the calling code if possible)
        float factor = sqrtf((float)size);

        // Use temporary variable directly
        float tmp = normcdf(from[id] * factor);

        // Store result in output array
        to[id] = (tmp - 0.5f) * 2.0f;
    }
}