#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RBMInputForwardAndStoreKernel(float *inputPtr, float *outputPtr, float *biasPtr, float *storePtr, bool applyBias, int thisLayerSize)
{
    // Calculate global index directly
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (i < thisLayerSize)
    {
        float result = inputPtr[i];
        if (applyBias)
            result += biasPtr[i];

        // Simultaneous writing to output and store
        outputPtr[i] = result;
        storePtr[i] = result;
    }
}