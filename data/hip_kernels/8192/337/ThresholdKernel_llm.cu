#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ThresholdKernel(float min, float max, int mode, float* input, float* output, int size, int count)
{
    int id = blockDim.x * blockIdx.y * gridDim.x
    + blockDim.x * blockIdx.x
    + threadIdx.x;

    __shared__ float delta;

    if(threadIdx.x == 0) {
        delta = (max - min) / count; // Compute delta once using shared memory
    }
    __syncthreads();

    if(id < size) {
        for (int i = 0; i < count; i++) {
            output[i * size + id] = 0.0f; // Initialize output using floating point number
        }

        int idx;
        float fidx = ((input[id] - min) / delta);

        switch (mode)
        {
        case 0: // consider values outside of the interval <min,max>
            idx = (int)floor(fmaxf(0.0f, fminf(fidx, count - 1))); // Use consistent fmaxf and fminf
            break;
        case 1: // strict threshold
            if (fidx < 0.0f || fidx >= count) {
                return;
            }
            idx = (int)fidx;
            break;
        }
        output[idx * size + id] = 1.0f; // Write to output only at valid position
    }
}