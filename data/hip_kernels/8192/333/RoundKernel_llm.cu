#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RoundKernel(float* input, float* output, int size)
{
    // Optimize grid index calculation
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    if(id < size)
    {
        // Use device function for rounding
        output[id] = __float2int_rn(input[id]);
    }
}