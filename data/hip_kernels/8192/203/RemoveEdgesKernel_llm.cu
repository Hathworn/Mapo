#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RemoveEdgesKernel(int *connection, int *age, int maxAge, int *activityFlag, float *winningFraction, int *winningCount, float *utility, float *localError, int *neuronAge, int maxCells)
{
    int threadId = blockDim.x * blockIdx.y * gridDim.x // rows preceding the current row in grid
                 + blockDim.x * blockIdx.x              // blocks preceding current block
                 + threadIdx.x;

    if (threadId < maxCells)
    {
        if (activityFlag[threadId] == 1)
        {
            neuronAge[threadId] += 1;

            int activeConnections = 0;
            for (int c = 0; c < maxCells; c++)
            {
                int connId = threadId * maxCells + c;
                if (connection[connId] == 1)
                {
                    // Simplified conditional check
                    if (age[connId] > maxAge)
                    {
                        connection[connId] = 0;
                        age[connId] = 0;
                    }
                    else
                    {
                        activeConnections++;
                    }
                }
            }

            if (activeConnections == 0)
            {
                activityFlag[threadId] = 0;
                localError[threadId] = 0.00f;
                neuronAge[threadId] = 0;
                winningFraction[threadId] = 0.00f;
                winningCount[threadId] = 0;
                utility[threadId] = 0.00f;
            }
        }
    }
}