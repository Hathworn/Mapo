#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FullyConnectedEstimateLearningRateKernel( float *weightLearningRatePtr, float *biasLearningRatePtr, float *avgWeightGradPtr, float *avgBiasGradPtr, float *avgWeightGradVarPtr, float *avgBiasGradVarPtr, float *avgWeightGradCurvePtr, float *avgBiasGradCurvePtr, float *avgWeightGradCurveVarPtr, float *avgBiasGradCurveVarPtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize )
{
    // Calculate global thread index
    int j = blockDim.x * blockIdx.y * gridDim.x
            + blockDim.x * blockIdx.x
            + threadIdx.x;

    if (j >= thisLayerSize || dropoutMaskPtr[j])
        return;

    int index = j;
    for (int i = 0; i < prevLayerSize; i++)
    {
        // Calculate weight learning rate: precompute repeated terms
        float grad = avgWeightGradPtr[index];
        weightLearningRatePtr[index] = (avgWeightGradCurvePtr[index] / avgWeightGradCurveVarPtr[index]) * ((grad * grad) / avgWeightGradVarPtr[index]);
        index += thisLayerSize;
    }

    // Calculate bias learning rate: precompute repeated terms
    float biasGrad = avgBiasGradPtr[j];
    biasLearningRatePtr[j] = (avgBiasGradCurvePtr[j] / avgBiasGradCurveVarPtr[j]) * ((biasGrad * biasGrad) / avgBiasGradVarPtr[j]);
}