#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ChangeInputWeightsKernel( float *inputWeights, float *inputWeightDeltas, float *outputWeights, float *outputDeltas, float *inputWeightRTRLDerivatives,  float trainingRate, float momentum )
{
    int weightId = blockDim.x * blockIdx.y * gridDim.x   // Calculate weightId considering preceding blocks and threads
                 + blockDim.x * blockIdx.x
                 + threadIdx.x;

    if (weightId < D_HIDDEN_UNITS * D_INPUT_UNITS)
    {
        float gradient = 0;

        // Implement parallel reduction pattern
        for (int i = 0; i < D_OUTPUT_UNITS; i++)
        {
            float sum = 0;
            #pragma unroll
            for (int j = 0; j < D_HIDDEN_UNITS; j++)
            {
                sum += outputWeights[i * D_HIDDEN_UNITS + j] * inputWeightRTRLDerivatives[j * D_HIDDEN_UNITS * D_INPUT_UNITS + weightId];
            }

            gradient += outputDeltas[i] * sum;
        }

        // Optimize weight update calculation
        float weightDelta = trainingRate * gradient + momentum * inputWeightDeltas[weightId];
        inputWeightDeltas[weightId] = weightDelta;
        inputWeights[weightId] += weightDelta;
    }
}