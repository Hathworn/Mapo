#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GaussianSamplePrior(float* input, int inputCount, float* mins, float* maxes, float* randomUniform)
{
    // Optimization: Use a 1D grid and 1D block for simpler indexing
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < inputCount)
    {
        float diff = maxes[i] - mins[i];
        input[i] = randomUniform[i] * diff + mins[i];
    }
}