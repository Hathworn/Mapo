#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeDistanceKernel( float *symbolVectors, float *inputVector, float *distance, int symbolSize, int symbols )
{
    int symbolId = blockIdx.x * blockDim.x + threadIdx.x; // Simplify thread index calculation
    
    if(symbolId < symbols)
    {
        float sum = 0.0f; // Initialize the sum variable
        float *symbolVector = symbolVectors + symbolId * symbolSize; // Pointer to the current symbol vector

        for(int i = 0; i < symbolSize; i++)
        {
            sum += symbolVector[i] * inputVector[i]; // Load and multiply elements directly
        }
        
        distance[symbolId] = sum; // Store the computed distance
    }
}