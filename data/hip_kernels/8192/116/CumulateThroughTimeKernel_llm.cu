#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CumulateThroughTimeKernel(float* memoryBlocks, int count, int sequenceLength)
{
    // Calculate memory index for this thread
    int memoryIdx = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    
    // Check if memory index is within bounds
    if (memoryIdx < count)
    {
        // Load initial value outside loop for better performance
        float cumulativeSum = memoryBlocks[memoryIdx];

        // Loop through sequenceTime and accumulate values
        for (size_t i = 1; i < sequenceLength; i++)
        {
            int memoryBlockOffset = i * count;
            cumulativeSum += memoryBlocks[memoryBlockOffset + memoryIdx];
        }

        // Store cumulative sum back to memoryBlocks
        memoryBlocks[memoryIdx] = cumulativeSum;
    }
}