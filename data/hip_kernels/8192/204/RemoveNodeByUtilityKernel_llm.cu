#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RemoveNodeByUtilityKernel(int *connectionMatrix, int *connectionAge, int *activityFlag, float *utility, float utilityConstant, float *localError, int *neuronAge, float *winningFraction, int *winningCount, float maxError, int maxCells)
{
    int threadId = blockDim.x * blockIdx.y * gridDim.x  // calculate global thread id
                + blockDim.x * blockIdx.x
                + threadIdx.x;

    // Check if threadId is within valid range
    if (threadId < maxCells)
    {
        // Check if current node is active
        if (activityFlag[threadId] == 1 && utility[threadId] > 0.00f)
        {
            // Check condition to remove node by utility
            if (maxError / utility[threadId] > utilityConstant)
            {
                activityFlag[threadId] = 0;
                localError[threadId] = 0.00f;
                neuronAge[threadId] = 0;
                winningFraction[threadId] = 0.00f;
                winningCount[threadId] = 0;
                utility[threadId] = 0.00f;

                // Unroll loop for performance
                #pragma unroll
                for (int n = 0; n < maxCells; n++)
                {
                    connectionMatrix[threadId * maxCells + n] = 0;
                    connectionAge[threadId * maxCells + n] = 0;
                    connectionMatrix[n * maxCells + threadId] = 0;
                    connectionAge[n * maxCells + threadId] = 0;
                }
            }
        }
    }
}