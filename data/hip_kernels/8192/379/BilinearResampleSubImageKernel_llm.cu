#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BilinearResampleSubImageKernel(float *input, float *output, float* subImageDefs, bool safeBounds, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
    // Calculate the global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = outputWidth * outputHeight;

    // Ensure the thread is within bounds
    if (id < size)
    {
        float subImgCX = subImageDefs[0];
        float subImgCY = subImageDefs[1];
        float subImgDiameter = subImageDefs[2];

        int maxDiameter = min(inputWidth - 1, inputHeight - 1);
        int diameterPix = (int)(subImgDiameter * maxDiameter);

        diameterPix = max(1, diameterPix);
        diameterPix = min(maxDiameter, diameterPix);

        int subImgX = (int)(inputWidth * (subImgCX + 1) * 0.5f) - diameterPix / 2;
        int subImgY = (int)(inputHeight * (subImgCY + 1) * 0.5f) - diameterPix / 2;

        if (safeBounds)
        {
            subImgX = max(subImgX, 1);
            subImgY = max(subImgY, 1);
            subImgX = min(subImgX, inputWidth - diameterPix - 1);
            subImgY = min(subImgY, inputHeight - diameterPix - 1);
        }

        int px = id % outputWidth;
        int py = id / outputWidth;

        float xRatio = (float)(diameterPix - 1) / (outputWidth - 1);
        float yRatio = (float)(diameterPix - 1) / (outputHeight - 1);

        int x = (int)(xRatio * px);
        int y = (int)(yRatio * py);

        // Ensure the computed coordinates are within the input image bounds
        if (x + subImgX >= 0 && y + subImgY >= 0 &&
            x + subImgX < inputWidth && y + subImgY < inputHeight)
        {
            float xDist = (xRatio * px) - x;
            float yDist = (yRatio * py) - y;

            float topLeft = input[(y + subImgY) * inputWidth + x + subImgX];
            float topRight = input[(y + subImgY) * inputWidth + x + subImgX + 1];
            float bottomLeft = input[(y + subImgY + 1) * inputWidth + x + subImgX];
            float bottomRight = input[(y + subImgY + 1) * inputWidth + x + subImgX + 1];

            // Compute the bilinear interpolation
            output[py * outputWidth + px] =
                topLeft * (1 - xDist) * (1 - yDist) +
                topRight * xDist * (1 - yDist) +
                bottomLeft * yDist * (1 - xDist) +
                bottomRight * xDist * yDist;
        }
    }
}