#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ApplyEyeMovement(float* currentEye, float* stats, float moveFactor, float scaleFactor, float scaleBase)
{
    // Read sumWeights and check its condition
    float sumWeights = stats[4];
    if (sumWeights > 0)
    {
        // Optimize by storing intermediate calculations in variables
        float clampedX = fmaxf(fminf(moveFactor * stats[0], 1.0f), -1.0f);
        float clampedY = fmaxf(fminf(moveFactor * stats[1], 1.0f), -1.0f);

        float combinedVariance = (stats[2] + stats[3]) * 0.5f;
        float variance = sqrtf(combinedVariance);

        float clampedZ = fmaxf(fminf(variance * scaleFactor + scaleBase, 1.0f), 0.0f);

        // Assign results
        currentEye[0] = clampedX;
        currentEye[1] = clampedY;
        currentEye[2] = clampedZ;
    }
    else
    {
        // Default condition values
        currentEye[0] = 0.0f;
        currentEye[1] = 0.0f;
        currentEye[2] = 1.0f;
    }
}