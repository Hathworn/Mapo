#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DropoutMaskKernel(float *dropoutMaskPtr, float dropout, int inputSize)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Ensure within bounds
    if (i < inputSize)
    {
        // Use ternary operator for condition assignment
        dropoutMaskPtr[i] = dropout > dropoutMaskPtr[i] ? 1.0f : 0.0f;
    }
}