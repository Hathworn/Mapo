#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LSTMCellInputGradientKernelBPTT(float *input, float *previousOutput, float *cellInputDeltas, float *cellInputWeightGradient, int inputCount, int previousOutputCount, int cellsPerBlock) 
{
    // Calculate weight index for this thread
    int weightId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    int weightsPerCell = inputCount + previousOutputCount + 1;

    // Ensure weightId is within bounds
    if (weightId >= weightsPerCell * previousOutputCount) return;

    int fromId = weightId % weightsPerCell;
    int toId = weightId / weightsPerCell;

    // Use boolean values to conditionally select which input to use
    int isFromInputUnit = (fromId < inputCount);
    int isFromPreviousOutputUnit = (fromId >= inputCount && fromId < inputCount + previousOutputCount);
    int isFromBiasUnit = (fromId == inputCount + previousOutputCount);

    // Use conditional operator for cleaner assignment
    float inputFromWeight = isFromInputUnit * input[fromId] +
                            isFromPreviousOutputUnit * previousOutput[fromId - inputCount] +
                            isFromBiasUnit * 1.0f;

    // Calculate the gradient
    cellInputWeightGradient[weightId] = cellInputDeltas[toId] * inputFromWeight;
}