#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RecurrentWeightsRTRLDerivativesKernel( float *previousHiddenActivations, float *hiddenActivationDerivatives, float *recurrentWeights, float *recurrentWeightRTRLDerivatives, float *previousRecurrentWeightRTRLDerivatives )
{
    int partialId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    // Optimize loop calculations
    int totalHiddenUnitsSquared = D_HIDDEN_UNITS * D_HIDDEN_UNITS;
    if (partialId < totalHiddenUnitsSquared * D_HIDDEN_UNITS)
    {
        int unitId = partialId / totalHiddenUnitsSquared;
        int weightId = partialId % totalHiddenUnitsSquared;
        int to = weightId / D_HIDDEN_UNITS;
        int from = weightId % D_HIDDEN_UNITS;

        float sum = 0.0f;
        for (int i = 0; i < D_HIDDEN_UNITS; i++)
        {
            sum += recurrentWeights[unitId * D_HIDDEN_UNITS + i] * previousRecurrentWeightRTRLDerivatives[i * totalHiddenUnitsSquared + weightId];
        }

        // Optimize condition evaluation using ternary operation
        recurrentWeightRTRLDerivatives[partialId] = hiddenActivationDerivatives[unitId] * ((unitId == to ? previousHiddenActivations[from] : 0.0f) + sum);
    }
}