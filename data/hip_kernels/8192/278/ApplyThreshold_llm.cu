#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ApplyThreshold(float* probabilitiesInputs, float* binaryOutput, float* probability, int count) {
    // Calculate global thread ID with optimized grid setup
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the current id is within bounds
    if (id < count) {
        // Use ternary operator for branch reduction
        binaryOutput[id] = (probabilitiesInputs[id] < probability[0]) ? 0.0f : 1.0f;
    }
}