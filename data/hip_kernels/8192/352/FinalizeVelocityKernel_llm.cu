#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FinalizeVelocityKernel(float* velocities, float* globalFlow, int inputWidth, int inputHeight)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation
    int size = inputWidth * inputHeight;

    if (id < size)
    {
        // Use shared memory for globalFlow calculation if applicable
        __shared__ float globalFlowShared[2];

        if (threadIdx.x == 0) {
            globalFlowShared[0] = globalFlow[0];
            globalFlowShared[1] = globalFlow[1];
        }
        __syncthreads();

        float globalFlowL = sqrtf(globalFlowShared[0] * globalFlowShared[0] + globalFlowShared[1] * globalFlowShared[1]);
        float velocityL = sqrtf(velocities[id] * velocities[id]  + velocities[size + id] * velocities[size + id]);

        if (globalFlowL > 0 && velocityL > 0) {

            float dot = (globalFlowShared[0] * velocities[id] + globalFlowShared[1] * velocities[size + id]) / (globalFlowL * velocityL);

            if (dot > 0.7) {
                velocities[id] = 0;
                velocities[size + id] = 0;
            }
        }
    }
}