#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void IncrementConnectionAgeKernel( int cell, int *connection, int *age, int maxCells )
{
    // Calculate thread index in a more efficient way
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if threadId is within bounds
    if(threadId < maxCells)
    {
        // Optimize memory access by using shared memory
        __shared__ int shared_connection[blockDim.x];
        __shared__ int shared_age[blockDim.x];

        // Load data into shared memory
        shared_connection[threadIdx.x] = connection[cell * maxCells + threadId];
        shared_age[threadIdx.x] = age[cell * maxCells + threadId];

        __syncthreads();

        // Process data using shared memory
        if(shared_connection[threadIdx.x] == 1)
        {
            shared_age[threadIdx.x] += 1;
            age[threadId * maxCells + cell] += 1;
        }

        // Store results back to global memory
        age[cell * maxCells + threadId] = shared_age[threadIdx.x];
    }
}