#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ChangeRecurrentWeightsKernel(float *recurrentWeights, float *recurrentWeightDeltas, float *outputWeights, float *outputDeltas, float *recurrentWeightRTRLDerivatives, float trainingRate, float momentum)
{
    int weightId = blockDim.x * blockIdx.y * gridDim.x // Calculate global thread ID
                  + blockDim.x * blockIdx.x
                  + threadIdx.x;

    if (weightId < D_HIDDEN_UNITS * D_HIDDEN_UNITS)
    {
        float gradient = 0;
        // Unroll loop to enhance performance
        for (int i = 0; i < D_OUTPUT_UNITS; i++)
        {
            float sum = 0;
            for (int j = 0; j < D_HIDDEN_UNITS; j += 4) // Loop unrolling by a factor of 4
            {
                sum += outputWeights[i * D_HIDDEN_UNITS + j] * recurrentWeightRTRLDerivatives[j * D_HIDDEN_UNITS * D_HIDDEN_UNITS + weightId];
                if (j + 1 < D_HIDDEN_UNITS)
                    sum += outputWeights[i * D_HIDDEN_UNITS + j + 1] * recurrentWeightRTRLDerivatives[(j + 1) * D_HIDDEN_UNITS * D_HIDDEN_UNITS + weightId];
                if (j + 2 < D_HIDDEN_UNITS)
                    sum += outputWeights[i * D_HIDDEN_UNITS + j + 2] * recurrentWeightRTRLDerivatives[(j + 2) * D_HIDDEN_UNITS * D_HIDDEN_UNITS + weightId];
                if (j + 3 < D_HIDDEN_UNITS)
                    sum += outputWeights[i * D_HIDDEN_UNITS + j + 3] * recurrentWeightRTRLDerivatives[(j + 3) * D_HIDDEN_UNITS * D_HIDDEN_UNITS + weightId];
            }

            gradient += outputDeltas[i] * sum;
        }

        float weightDelta = trainingRate * gradient + momentum * recurrentWeightDeltas[weightId];
        recurrentWeightDeltas[weightId] = weightDelta;
        recurrentWeights[weightId] += weightDelta;
    }
}