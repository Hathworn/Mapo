#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void IntervalToBinaryVector(float input, float* outputs, int steps)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation

    if (id < steps)
    {
        float fraction = 1.0f / steps;
        // Optimized condition check with enhanced readability
        outputs[id] = (input >= id * fraction) && (input < (id + 1) * fraction);
    }
}