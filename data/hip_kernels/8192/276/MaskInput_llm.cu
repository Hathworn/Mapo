#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel: Use 2D grid indices and better index calculation
__global__ void MaskInput(float* image, float* mask, float* maskedValues, float* output, int count) {
    // Calculate unique global thread index efficiently
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    if (id < count) {
        // Perform mask input operation
        float maskValue = mask[id];
        output[id] = image[id] * maskValue + maskedValues[id] * (1.0f - maskValue);
    }
}