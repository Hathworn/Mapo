#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FullyConnectedAdjustMemoryKernel( float *weightsGradPtr, float *biasGradPtr, float *weightGradCurvePtr, float *biasGradCurvePtr, float *avgWeightGradPtr, float *avgBiasGradPtr, float *avgWeightGradVarPtr, float *avgBiasGradVarPtr, float *avgWeightGradCurvePtr, float *avgBiasGradCurvePtr, float *avgWeightGradCurveVarPtr, float *avgBiasGradCurveVarPtr, float *weightMemorySizePtr, float *biasMemorySizePtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize )
{
    // Calculate j directly using linear index for better readability
    int j = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    if (j < thisLayerSize)
    {
        if (!dropoutMaskPtr[j])
        {
            int index = j;
            for (int i = 0; i < prevLayerSize; i++)
            {
                // Check for weight outliers: simplify through index reuse and reduce recalculations
                float weightGradDiff = weightsGradPtr[index] - avgWeightGradPtr[index];
                float weightGradCurveDiff = weightGradCurvePtr[index] - avgWeightGradCurvePtr[index];
                
                float weightGradVarSqrt = sqrtf(avgWeightGradVarPtr[index]);
                float weightGradCurveVarSqrt = sqrtf(avgWeightGradCurveVarPtr[index]);

                float weightVarianceTerm = weightGradVarSqrt - avgWeightGradPtr[index] * avgWeightGradPtr[index];
                float weightCurveVarianceTerm = weightGradCurveVarSqrt - avgWeightGradCurveVarPtr[index] * avgWeightGradCurveVarPtr[index];
                
                if (abs(weightGradDiff) > 2 * weightVarianceTerm || abs(weightGradCurveDiff) > 2 * weightCurveVarianceTerm)
                {
                    weightMemorySizePtr[index] = 2.2f;  // simplified; chosen Adasecant method
                }
                
                index += thisLayerSize;
            }

            // Check for bias outliers: reuse calculations similar to weight section
            float biasGradDiff = biasGradPtr[j] - avgBiasGradPtr[j];
            float biasGradCurveDiff = biasGradCurvePtr[j] - avgBiasGradCurvePtr[j];
            
            float biasGradVarSqrt = sqrtf(avgBiasGradVarPtr[j]);
            float biasGradCurveVarSqrt = sqrtf(avgBiasGradCurveVarPtr[j]);

            float biasVarianceTerm = biasGradVarSqrt - avgBiasGradPtr[j] * avgBiasGradPtr[j];
            float biasCurveVarianceTerm = biasGradCurveVarSqrt - avgBiasGradCurveVarPtr[j] * avgBiasGradCurveVarPtr[j];

            if (abs(biasGradDiff) > 2 * biasVarianceTerm || abs(biasGradCurveDiff) > 2 * biasCurveVarianceTerm)
            {
                biasMemorySizePtr[j] = 2.2f;  // simplified; chosen Adasecant method
            }
        }
    }
}