#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PolynomialFunctionKernel_Double(float a3, float a2, float a1, float a0, double* input, double* output, int size)
{
    // Improved 1D block and grid indexing for better performance
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < size)
    {
        // Minimize redundant memory access by storing result in a register
        double x = input[id];
        output[id] = fma(fma(fma(a3, x, a2) * x, a1, a0), x, a0); // Utilize FMA for better accuracy and performance
    }
}