#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PrepareMeanStdDev(float* input, float* delta, int imageWidth, int imageHeight)
{
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    int size = imageWidth * imageHeight;

    if (id < size)
    {
        float invWidth = 2.0f / imageWidth;
        float invHeight = 2.0f / imageHeight;
        
        // Calculate pixel positions
        int px = id % imageWidth;
        int py = id / imageWidth;
        float pixPosX = invWidth * px - 1.0f;
        float pixPosY = invHeight * py - 1.0f;
        
        // Load input data
        float inputVal = input[id];
        
        // Mean sum
        delta[id] = inputVal * pixPosX;       // x component
        delta[id + size] = inputVal * pixPosY; // y component
        
        // Variance sum
        delta[id + 2 * size] = inputVal * pixPosX * pixPosX; // x^2 component
        delta[id + 3 * size] = inputVal * pixPosY * pixPosY; // y^2 component
    }
}