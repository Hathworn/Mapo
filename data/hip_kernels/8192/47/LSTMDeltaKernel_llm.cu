#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LSTMDeltaKernel(float *cellStateErrors, float *outputGateDeltas, float *cellStates,
                                float *outputGateActivations, float *outputGateActivationDerivatives,
                                float *deltas, int cellCount, int cellsPerBlock)
{
    // Optimize index calculation for better readability and reusability
    int memoryBlockId = blockIdx.x + blockIdx.y * gridDim.x;

    if (memoryBlockId * cellsPerBlock < cellCount)
    {
        float outputGateDeltaSum = 0.0;
        // Use registers to minimize memory accesses
        float sharedOutputGateActivation = outputGateActivations[memoryBlockId];
        float sharedOutputGateActivationDerivative = outputGateActivationDerivatives[memoryBlockId];
        
        for (int i = 0; i < cellsPerBlock; ++i)
        {
            int cellId = memoryBlockId * cellsPerBlock + i;
            if (cellId < cellCount)
            {
                float delta = deltas[cellId];
                cellStateErrors[cellId] = sharedOutputGateActivation * delta;
                outputGateDeltaSum += cellStates[cellId] * delta;
            }
        }
        
        // Reduce redundant computations by using shared variables
        outputGateDeltas[memoryBlockId] = sharedOutputGateActivationDerivative * outputGateDeltaSum;
    }
}