#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DrawRgbaTextureKernel2DBlock(float *target, int targetWidth, int targetHeight, int inputX, int inputY, float *texture, int textureWidth, int textureHeight)
{
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = threadIdx.y * blockDim.x + threadIdx.x;
    int id = blockDim.x * blockDim.y * blockId + threadId;

    int idTextureRgb = blockIdx.y;
    int idTexturePixel = threadId;
    int idTextureY = blockIdx.x * blockDim.y + threadIdx.y;
    int idTextureX = threadIdx.x;

    if (idTextureRgb < 3) // 3 channels that we will write to
    {
        // the texture is in BGR format, we want RGB
        idTextureRgb = (idTextureRgb == 0) ? 2 : (idTextureRgb == 2) ? 0 : idTextureRgb;

        // if the texture pixel offset by inputX, inputY, lies inside the target
        if (idTextureX + inputX < targetWidth &&
            idTextureX + inputX >= 0 &&
            idTextureY + inputY < targetHeight &&
            idTextureY + inputY >= 0)
        {
            int targetPixels = targetWidth * targetHeight;
            int texturePixels = textureWidth * textureHeight;
            int tIndex = targetPixels * idTextureRgb + targetWidth * (idTextureY + inputY) + (idTextureX + inputX);
            int aIndex = idTexturePixel + 3 * texturePixels; // the A component of the texture
            float a = texture[aIndex];
            target[tIndex] = target[tIndex] * (1.0f - a) + a * texture[id];
        }
    }
}