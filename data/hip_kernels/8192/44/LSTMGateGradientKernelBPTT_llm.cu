#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LSTMGateGradientKernelBPTT(float *input, float *previousOutput, float *cellStates, float *inputGateDeltas, float *forgetGateDeltas, float *outputGateDeltas, float* outputGateWeightGradient, float* inputGateWeightGradient, float* forgetGateWeightGradient, int inputCount, int previousOutputCount, int cellsPerBlock)
{
    // Calculate weightId more neatly
    int weightId = blockIdx.x + blockIdx.y * gridDim.x * blockDim.x + threadIdx.x;

    int weightsPerGate = inputCount + previousOutputCount + cellsPerBlock + 1;

    if (weightId < weightsPerGate * previousOutputCount / cellsPerBlock)
    {
        int fromId = weightId % weightsPerGate;
        int toId = weightId / weightsPerGate;

        bool isFromInputUnit = fromId < inputCount;
        bool isFromPreviousOutputUnit = fromId >= inputCount && fromId < inputCount + previousOutputCount;
        bool isPeephole = fromId >= inputCount + previousOutputCount && fromId < inputCount + previousOutputCount + cellsPerBlock;
        bool isFromBiasUnit = fromId == inputCount + previousOutputCount + cellsPerBlock;

        float inputFromWeight = 0.0f;
        if (isFromInputUnit) {
            inputFromWeight = input[fromId];
        } else if (isFromPreviousOutputUnit) {
            inputFromWeight = previousOutput[fromId - inputCount];
        } else if (isPeephole) {
            inputFromWeight = cellStates[toId * cellsPerBlock + (fromId - inputCount - previousOutputCount)];
        } else if (isFromBiasUnit) {
            inputFromWeight = 1.0f;
        }

        // Optimization: reduce redundant multiplication operations
        outputGateWeightGradient[weightId] = outputGateDeltas[toId] * inputFromWeight;
        inputGateWeightGradient[weightId] = inputGateDeltas[toId] * inputFromWeight;
        forgetGateWeightGradient[weightId] = forgetGateDeltas[toId] * inputFromWeight;
    }
}