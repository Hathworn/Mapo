#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FullyConnectedUpdateParametersKernel( float *weightPtr, float *biasPtr, float *weightLearningRatePtr, float *biasLearningRatePtr, float *weightGradientPtr, float *biasGradientPtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize )
{
    // Calculate neuron id (j) in this layer
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure the current thread is within the bounds
    if (j < thisLayerSize)
    {
        if (!dropoutMaskPtr[j])
        {
            // Prefetch common offset calculations
            float *wPtr = weightPtr + j;
            float *wLrPtr = weightLearningRatePtr + j;
            float *wGPtr = weightGradientPtr + j;

            // Update weights using stride pointer arithmetic
            for (int i = 0; i < prevLayerSize; i++, wPtr += thisLayerSize, wLrPtr += thisLayerSize, wGPtr += thisLayerSize) 
            {
                *wPtr -= *wLrPtr * *wGPtr;
            }

            // Update bias
            biasPtr[j] -= biasLearningRatePtr[j] * biasGradientPtr[j];
        }
    }
}