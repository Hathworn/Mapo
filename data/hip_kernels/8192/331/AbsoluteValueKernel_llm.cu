#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AbsoluteValueKernel(float* input, float* output, int size)
{
    // Calculate global thread ID in a more conventional manner
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Conditional check remains unchanged
    if (id < size)
    {
        output[id] = fabsf(input[id]);
    }
}