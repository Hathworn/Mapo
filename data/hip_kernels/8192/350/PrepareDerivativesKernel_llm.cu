#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PrepareDerivativesKernel(float* input, float* lastInput, float* derivatives, int inputWidth, int inputHeight)
{
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    int size = inputWidth * inputHeight;

    if (id < size) 
    {
        float mul = 100000.0f;
        // Load current and last input values
        float currInput = input[id];
        float lastInputVal = lastInput[id];

        // Calculate first-order derivatives
        float I_x = mul * derivatives[id];
        float I_y = mul * derivatives[size + id];
        float input_dt = mul * (currInput - lastInputVal);

        // Update last input value
        lastInput[id] = currInput;

        // Calculate second-order derivatives
        derivatives[2 * size + id] = I_x * I_y;
        derivatives[3 * size + id] = I_x * input_dt;
        derivatives[4 * size + id] = I_y * input_dt;
        derivatives[id] = I_x * I_x;
        derivatives[size + id] = I_y * I_y;
    }
}