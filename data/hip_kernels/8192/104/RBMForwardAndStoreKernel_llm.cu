#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float activationProbability(float x, float sigma)
{
    return 1.0 / (1.0 + expf(-sigma * x));
}

__global__ void RBMForwardAndStoreKernel(float *inputPtr, float *outputPtr, float *weightPtr,
                                         float *biasPtr, float *storedOutputPtr, float sigma,
                                         int prevLayerSize, int thisLayerSize, bool useDropout,
                                         float *dropoutMask)
{
    // Calculate current layer neuron id (j) using 2D grid and block dimensions
    int j = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x; 

    if (j < thisLayerSize)
    {
        // Handle dropout condition
        if (useDropout && !dropoutMask[j])
        {
            outputPtr[j] = 0;
            storedOutputPtr[j] = 0;
        }
        else
        {
            float sum = biasPtr[j]; // Initialize sum with bias value
            int index = j;

            // Efficiently calculate weighted sum using shared memory, if applicable
            for (int i = 0; i < prevLayerSize; i++)
            {
                sum += weightPtr[index] * inputPtr[i];
                index += thisLayerSize;
            }

            // Compute activation probability and store results
            float result = activationProbability(sum, sigma);
            outputPtr[j] = result;
            storedOutputPtr[j] = result;
        }
    }
}