#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AbsKernel_optimized(const float * A, int Acount, int Acols, float * out0, int out0count)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop through elements with stride for better occupancy
    for (int i = id; i < out0count; i += stride)
    {
        out0[i] = fabsf(A[i]);
    }
}