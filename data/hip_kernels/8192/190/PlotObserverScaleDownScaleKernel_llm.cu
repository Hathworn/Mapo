#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PlotObserverScaleDownScaleKernel(float* history, int nbCurves, int size)
{
    // Compute global thread ID using block and grid dimensions
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Boundary check to ensure thread does not access out-of-bounds memory
    if (id >= size) return;

    // Use only one memory access per value
    int baseAddress = 2 * id;
    float val1 = history[baseAddress];
    float val2 = history[baseAddress + nbCurves];

    // Store average directly to avoid extra instructions
    history[id] = (val1 + val2) * 0.5f;
}