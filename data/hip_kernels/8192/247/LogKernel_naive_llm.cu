#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LogKernel_optimized(const float *A, int Acount, int Acols, float *out0, int out0count)
{
    // Calculate unique global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop to handle more elements per thread and improve data parallelism
    for(int i = id; i < out0count; i += blockDim.x * gridDim.x)
    {
        out0[i] = logf(A[i]); // Use logf for increased performance with single-precision floats
    }
}