#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RBMDropoutMaskKernel(float *maskPtr, float dropout, int thisLayerSize)
{
    int index = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x; // Flattened linear index calculation

    if (index < thisLayerSize)
    {
        maskPtr[index] = dropout < maskPtr[index]; // Apply dropout mask
    }
}