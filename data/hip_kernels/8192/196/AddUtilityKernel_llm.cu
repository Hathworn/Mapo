#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AddUtilityKernel(int s1, int s2, float *distance, float *utility)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation

    if(threadId == 0) // Only the first thread performs the operation
    {
        utility[s1] += distance[s2] - distance[s1];
    }
}