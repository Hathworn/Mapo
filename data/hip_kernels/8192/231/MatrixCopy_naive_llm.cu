#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixCopy_optimized(const float *A, int Acount, int Acols, float *out0, int out0count)
{
    // Calculate global thread index for a 1D grid of 1D blocks
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform copy only if the thread index is within bounds
    if (id < out0count)
    {
        out0[id] = A[id];
    }
}