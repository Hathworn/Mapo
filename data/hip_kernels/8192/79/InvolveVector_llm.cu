#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized InvolveVector kernel function
__global__ void InvolveVector(float* input, float* output, int inputSize)
{
    // Calculate global thread ID using a more efficient method
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < inputSize) // Ensure the thread is within bounds
    {
        if (threadId == 0)
        {
            output[0] = input[0]; // Directly set the first element
        }
        else if (threadId < inputSize) // Reverse the rest
        {
            output[threadId] = input[inputSize - threadId];
        }
    }
}