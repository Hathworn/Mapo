#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UdpateEnergyTerm_movement(float* energy, int energy_dim, int nPatches, float *desc, int desc_dim, int id_desc_move) {

    // Calculate unique global index using grid and block dimensions
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    
    // Check within bounds
    if (id < energy_dim * nPatches) {
    
        // Compute indices corresponding to dimensions and patches
        int idDim   = id % energy_dim;
        int idPatch = id / energy_dim;

        // Directly update energy with movement condition
        if (idDim == 1) {
            // Update energy value corresponding to movement
            energy[id] = -desc[idPatch * desc_dim + id_desc_move];
        }
    }
}