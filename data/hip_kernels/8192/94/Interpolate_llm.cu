#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Interpolate(float* input1, float* input2, float* output, float weight, int inputSize)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread ID calculation

    if(threadId < inputSize)
    {
        float value1 = input1[threadId];
        float value2 = input2[threadId];

        // Use ternary operators to streamline conditional logic
        output[threadId] = (weight <= 0) ? value1 : ((weight >= 1) ? value2 : ((1 - weight) * value1 + weight * value2));
    }
}