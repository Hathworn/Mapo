#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InvertValuesKernel(float *input, float* outputs, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified ID computation

    if(id < size)
    {
        outputs[id] = 1.00f - input[id];
    }
}