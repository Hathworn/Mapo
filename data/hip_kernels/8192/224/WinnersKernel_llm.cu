#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void WinnersKernel( float *winner, float *vertexData, int vertexOffset, float *pointsCoordinates, float cubeSize, int maxCells )
{
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    // Use registers for frequently used calculations
    float side = 1.2f * cubeSize;
    float halfSize = 0.5f * side;

    if(threadId < maxCells && winner[threadId] == 1.00f)
    {
        // Use temporary variables to store repeated expressions
        float *point = &pointsCoordinates[threadId * 3];
        float x = point[0];
        float y = point[1];
        float z = point[2];

        // Reduce repeated calculations and improve memory access pattern
        float *vData = &vertexData[vertexOffset];
        vData[0] = x - halfSize; vData[1] = y - halfSize; vData[2] = z - halfSize;
        vData[3] = x - halfSize; vData[4] = y - halfSize; vData[5] = z + halfSize;
        vData[6] = x + halfSize; vData[7] = y - halfSize; vData[8] = z + halfSize;
        vData[9] = x + halfSize; vData[10] = y - halfSize; vData[11] = z - halfSize;
    }
}