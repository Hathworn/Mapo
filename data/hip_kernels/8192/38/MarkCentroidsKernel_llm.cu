#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MarkCentroidsKernel( float *centroidCoordinates, float *visField, int imgWidth, int imgHeight, int centroids )
{
    // Calculate unique thread ID with optimized formula
    int threadId = blockIdx.x * blockDim.x + threadIdx.x + blockDim.x * blockIdx.y * gridDim.x;
    
    if (threadId < centroids)
    {
        // Load coordinates to local variables for better memory access
        int x = lrintf(centroidCoordinates[threadId * 2]);
        int y = lrintf(centroidCoordinates[threadId * 2 + 1]);

        // Mark centroid in visibility field
        visField[y * imgWidth + x] = -1.00f;
    }
}