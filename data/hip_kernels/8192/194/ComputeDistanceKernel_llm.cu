#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeDistanceKernel(int inputSize, float *distance, float *dimensionWeight, int maxCells, float *difference)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread ID calculation

    if (threadId < maxCells)
    {
        float sum = 0.0f;
        for (int i = 0; i < inputSize; ++i)
        {
            float value = difference[threadId * inputSize + i];
            sum += dimensionWeight[i] * value * value;
        }
        distance[threadId] = sqrtf(sum);
    }
}