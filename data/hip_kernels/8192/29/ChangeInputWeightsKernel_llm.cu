#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ChangeInputWeightsKernel(float *inputWeights, float *inputWeightDeltas, float *outputWeights, float *outputDeltas, float *inputWeightRTRLDerivatives, float trainingRate, float momentum)
{
    int weightId = blockDim.x * blockIdx.y * gridDim.x // rows preceding current row in grid
                 + blockDim.x * blockIdx.x             // blocks preceding current block
                 + threadIdx.x;

    if (weightId < D_HIDDEN_UNITS * D_INPUT_UNITS)
    {
        float gradient = 0.0f;

        // Use shared memory to store partial sums for faster access
        extern __shared__ float sharedSum[];
        for (int i = 0; i < D_OUTPUT_UNITS; i++)
        {
            sharedSum[threadIdx.x] = 0.0f;
            for (int j = 0; j < D_HIDDEN_UNITS; j++)
            {
                sharedSum[threadIdx.x] += outputWeights[i * D_HIDDEN_UNITS + j] *
                                          inputWeightRTRLDerivatives[j * D_HIDDEN_UNITS * D_INPUT_UNITS + weightId];
            }
            __syncthreads(); // Ensure all threads complete the computation

            gradient += outputDeltas[i] * sharedSum[threadIdx.x];
            __syncthreads(); // Ensure all threads read updated gradient
        }

        float weightDelta = trainingRate * gradient + momentum * inputWeightDeltas[weightId];
        inputWeightDeltas[weightId] = weightDelta;
        inputWeights[weightId] += weightDelta;
    }
}