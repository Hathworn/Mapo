#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//#define _SIZE_T_DEFINED

extern "C"
{
}

__global__ void ShuffleRGB(float* input, float* output, int size)
{
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x; // Simplified id calculation

    if (id < size)
    {
        int index = (id / 3) + (id % 3) * (size / 3); // Precomputed index
        output[index] = input[id];
    }
}