#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeEuklidianDistancesKernel(float *inputImg, int imgWidth, int imgHeight, float *centroidCoordinates, float *distanceMatrix, int centroids, int inputSize)
{
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x; // Calculate global thread ID

    if (threadId < inputSize)
    {
        int pointX = threadId % imgWidth;
        int pointY = threadId / imgWidth;

        float X = static_cast<float>(pointX);
        float Y = static_cast<float>(pointY);

        for (int c = 0; c < centroids; c++)
        {
            float centroidX = centroidCoordinates[c * 2];
            float centroidY = centroidCoordinates[c * 2 + 1];

            float distX = centroidX - X;
            float distY = centroidY - Y;
            
            // Use a single sqrtf call after the loop
            distanceMatrix[c * inputSize + threadId] = sqrtf(distX * distX + distY * distY);
        }
    }
}