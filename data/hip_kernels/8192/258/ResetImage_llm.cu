#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ResetImage(float* im, int size)
{
    // Calculate the global thread ID more efficiently
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop for larger arrays to ensure all elements get reset
    for (int idx = id; idx < size; idx += blockDim.x * gridDim.x) {
        im[idx] = 0;
    }
}