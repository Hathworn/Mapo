#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DrawMaskedColorKernel(float *target, int targetWidth, int targetHeight, int inputX, int inputY, float *textureMask, int textureWidth, int textureHeight, float r, float g, float b)
{
    // Calculate the global thread index
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    int texturePixels = textureWidth * textureHeight;
    int idTextureRgb = id / texturePixels;
    
    if (idTextureRgb < 3) // Process only RGB channels
    {
        int idTexturePixel = id % texturePixels; // Use modulo for cleaner code
        int idTextureY = idTexturePixel / textureWidth;
        int idTextureX = idTexturePixel % textureWidth;

        // Check bounds with input offsets
        if (idTextureX + inputX < targetWidth && idTextureX + inputX >= 0 &&
            idTextureY + inputY < targetHeight && idTextureY + inputY >= 0)
        {
            int targetIndex = (idTextureRgb * targetHeight + (idTextureY + inputY)) * targetWidth + (idTextureX + inputX);
            int alphaIndex = idTexturePixel + 3 * texturePixels; // A channel index
            float alpha = textureMask[alphaIndex];

            if (alpha > 0) // Apply mask
            {
                switch (idTextureRgb)
                {
                    case 0:
                        target[targetIndex] = r;
                        break;
                    case 1:
                        target[targetIndex] = g;
                        break;
                    case 2:
                        target[targetIndex] = b;
                        break;
                }
            }
        }
    }
}
```
