#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_getRow_FloatId_naive(const float * A, int Acount, int Acols, float * out0, int out0count, int out0cols, float row_id) {
    // Calculate 1D grid-stride loop index to increase parallelism
    int id = blockDim.x * blockIdx.x + threadIdx.x; // Simplified grid index calculation
    int stride = blockDim.x * gridDim.x;            // Determine stride for grid-stride loop
    
    for (int idx = id; idx < Acols; idx += stride) { // Use grid-stride loop for accessing matrix row
        out0[idx] = A[idx + (int)row_id * Acols];    // Copy target row elements to output
    }
}