#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CreateAndRefreshConnectionKernel(int s1, int s2, int *connection, int *age, int maxCells)
{
    // Compute the unique threadId for the thread in the 1D grid
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Only the first thread needs to perform the operation
    if (threadId == 0)
    {
        // Update the connection and age matrices
        int index1 = s1 * maxCells + s2;
        int index2 = s2 * maxCells + s1;
        connection[index1] = 1;
        age[index1] = 0;
        connection[index2] = 1;
        age[index2] = 0;
    }
}