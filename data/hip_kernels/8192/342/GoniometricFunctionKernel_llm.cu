#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GoniometricFunctionKernel(float* input, float* output, const int size, const int type)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;  // Simplified grid-stride calculation
    if(id < size) // Ensure execution only within bounds
    {   
        float value = input[id];  // Cache input value to reduce global memory access
        switch (type)
        {
            case 0:
                output[id] = sinf(value);
                break;
            case 1:
                output[id] = cosf(value);
                break;
            case 2:
                output[id] = tanf(value);
                break;
            case 3:
                output[id] = tanhf(value);
                break;
            case 4:
                output[id] = sinhf(value);
                break;
            case 5:
                output[id] = coshf(value);
                break;
            case 6:
                output[id] = asinf(value);
                break;
            case 7:
                output[id] = acosf(value);
                break;
            // Handling the case of atan2 separately due to its unique access pattern
            case 10:
                if (id * 2 + 1 < size) // Additional boundary check for index safety
                    output[id] = atan2f(input[2 * id], input[2 * id + 1]);
                break;
        }
    }
}