#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PolynomialFunctionKernel(float a3, float a2, float a1, float a0, float* input, float* output, int size)
{
    // Use a 1D block index calculation
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id < size)
    {
        float x = input[id];
        // Calculate the polynomial in one line
        output[id] = fmaf(fmaf(fmaf(a3, x, a2) * x, a1) * x, a0);
    }
}