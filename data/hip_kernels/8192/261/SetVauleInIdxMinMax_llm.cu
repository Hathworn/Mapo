#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SetVauleInIdxMinMax(float* vector, int id_min, int id_max, float value)
{
    // Calculate the global thread index more efficiently
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Use an efficient check to ensure the index is within bounds
    if (id_min <= id && id <= id_max) {
        vector[id] = value;
    }
}