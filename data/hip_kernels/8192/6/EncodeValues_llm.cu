#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void EncodeValuesInternal(float value, float& origin, float& dir, float& output, int squaredMode)
{
    if (squaredMode == 1)
    {
        // origin part:      o * (1 - t)^2
        output = (1 - fabs(value)) * (1 - fabs(value)) * origin;
        // direction part:   dir * (-t^2 + 2*t)
        output += (-value * value + 2 * fabs(value)) * dir;
    }
    else
    {
        // origin part:      o * (1 - t)
        output = (1 - fabs(value)) * origin;
        // direction part:   dir * t
        output += fabs(value) * dir;
    }
}

__global__ void EncodeValues(float* values, int numOfValues, float* output, int symbolSize, int squaredMode, float* dirX, float* dirY, float* negDirX, float* negDirY, float* originX, float* originY)
{
    int threadId = blockDim.x * blockIdx.y * gridDim.x // rows preceding current row in grid
                  + blockDim.x * blockIdx.x            // blocks preceding current block
                  + threadIdx.x;

    __shared__ float s_values[2];

    // Load shared values and clamp if squaredMode
    if (threadIdx.x < 2)
    {
        s_values[threadIdx.x] = squaredMode == 1 ? fmaxf(fminf(values[threadIdx.x], 1), -1) : values[threadIdx.x];
    }

    __syncthreads();

    // Return early if the threadId is out of bounds
    if (threadId >= symbolSize)
        return;

    // Optimize direction pointer selection by reducing ternary operations
    float* dirX_local = (s_values[0] > 0) ? dirX : negDirX;
    float* dirY_local = (s_values[1] > 0) ? dirY : negDirY;

    // Encode X dim
    EncodeValuesInternal(s_values[0], originX[threadId], dirX_local[threadId], output[threadId], squaredMode);

    // Encode Y dim if applicable
    if (numOfValues > 1)
    {
        EncodeValuesInternal(s_values[1], originY[threadId], dirY_local[threadId], output[threadId], squaredMode);
    }
}