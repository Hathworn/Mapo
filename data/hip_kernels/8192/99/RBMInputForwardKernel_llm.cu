#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RBMInputForwardKernel(float *inputPtr, float *outputPtr, float *biasPtr, bool applyBias, int thisLayerSize)
{
    // Use linear thread ID calculation
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < thisLayerSize)
    {
        // Directly store the computed result to output
        float result = inputPtr[i];
        if (applyBias)
            result += biasPtr[i];
        outputPtr[i] = result; // Correct storing of the result
    }
}