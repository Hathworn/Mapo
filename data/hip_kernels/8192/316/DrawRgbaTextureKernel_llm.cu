#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DrawRgbaTextureKernel(float *target, int targetWidth, int targetHeight, int inputX, int inputY, float *texture, int textureWidth, int textureHeight)
{
  int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

  int texturePixels = textureWidth * textureHeight;

  // Calculate texture RGB and pixel indices
  int idTextureRgb = id / texturePixels;
  int idTexturePixel = id % texturePixels;
  int idTextureY = idTexturePixel / textureWidth;
  int idTextureX = idTexturePixel % textureWidth;

  if (idTextureRgb < 3) // Only process RGB channels
  {
    // Switch from BGR to RGB
    idTextureRgb = (idTextureRgb == 0) ? 2 : (idTextureRgb == 2) ? 0 : 1;

    // Check if pixel position is within target boundaries
    if (idTextureX + inputX < targetWidth &&
        idTextureX + inputX >= 0 &&
        idTextureY + inputY < targetHeight &&
        idTextureY + inputY >= 0)
    {
      int tIndex = (idTextureRgb * targetWidth * targetHeight) + (targetWidth * (idTextureY + inputY)) + (idTextureX + inputX);
      int aIndex = idTexturePixel + 3 * texturePixels; // Alpha channel index
      float a = texture[aIndex];
      
      // Blend texture into target
      target[tIndex] = target[tIndex] * (1.0f - a) + a * texture[id];
    }
  }
}