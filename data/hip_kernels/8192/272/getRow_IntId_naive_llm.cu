#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getRow_IntId_optimized(const float * A, int row_id, float * out, int Acols) {
// Efficient calculation of global thread ID using gridDim.x
int id = blockIdx.x * blockDim.x + threadIdx.x;
// Ensure threads within bounds of Acols
if (id < Acols) {
    // Directly access row elements
    out[id] = A[id + row_id * Acols];
}
}