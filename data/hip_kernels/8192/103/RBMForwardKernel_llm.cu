#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float activationProbability(float x, float sigma)
{
    return 1.0 / (1.0 + expf(-sigma * x));
}

__global__ void RBMForwardKernel(float *inputPtr, float *outputPtr, float *weightPtr, float *biasPtr, float sigma, int prevLayerSize, int thisLayerSize, bool useDropoutMask, bool useDropout, float dropoutRate, float *dropoutMask)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Improved index calculation

    if (j < thisLayerSize)
    {
        if (useDropout && useDropoutMask && !dropoutMask[j])
        {
            outputPtr[j] = 0;
        }
        else
        {
            float sum = biasPtr[j]; // Combine bias addition within loop
            for (int i = 0; i < prevLayerSize; i++)
            {
                sum += weightPtr[j + i * thisLayerSize] * inputPtr[i]; // Calculate weighted sum
            }

            float result = activationProbability(sum, sigma);

            if (useDropout && !useDropoutMask && dropoutRate < 1)
                result /= dropoutRate;

            outputPtr[j] = result;
        }
    }
}