#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixCopy(float* in, float *out, int size)
{
    // Improved linear thread indexing
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a strided loop for potential better occupancy
    for (int i = id; i < size; i += blockDim.x * gridDim.x) {
        out[i] = in[i];
    }
}