#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void detectChanges(float* a, float* b, float* result, int size, float value)
{
    // Calculate global thread ID more efficiently
    int threadId = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    if(threadId < size)
    {
        float a_val = a[threadId];
        float b_val = b[threadId];

        // Use ternary operators for compact conditional assignments
        result[threadId] = (a_val > b_val) ? value : (a_val < b_val) ? -value : 0;
    }
}