#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ModuloKernel(float* input, int divisor, float* output, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified 1D grid index calculation
    if (id < size)
    {
        output[id] = fmodf(input[id], divisor);  // Use fmodf for floating-point numbers
    }
}