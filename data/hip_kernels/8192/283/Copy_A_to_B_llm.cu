#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Copy_A_to_B(float *A, float *B, int size) {
    // Calculate global thread ID in a simplified way
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        B[id] = A[id];
    }
}