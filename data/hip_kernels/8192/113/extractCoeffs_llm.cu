#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void extractCoeffs(const float *matrices, float *coeffArray, int savedCoeffs, int dimsize) {
    // Compute global thread ID
    int threadGlobalID = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
                         + threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;

    // Calculate offsets
    int offsetMatrix = threadGlobalID * dimsize * dimsize;
    int offsetCoeff = threadGlobalID * savedCoeffs;
    int coeffsLeft = savedCoeffs;
    int x_n = 1, y_n = 0;
    int x, y, numberinrow, tmp;

    // Initialize first coefficient
    coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)] = matrices[offsetMatrix];
    coeffsLeft -= 1;

    // Loop through coefficients
    while (coeffsLeft > 0) {
        // Determine number in row
        x = x_n;
        y = y_n;
        numberinrow = (x_n < dimsize - 1) ? x_n + 1 : x_n - (y_n - 1);

        bool isEven = numberinrow % 2 == 0;
        while (numberinrow > 0 && coeffsLeft > 0) {
            // Save coefficient
            coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)] = matrices[offsetMatrix + x + y * dimsize];
            numberinrow--;
            coeffsLeft--;

            // Swap x and y as needed
            tmp = x;
            x = y;
            y = tmp;
            if (!isEven) {
                x--;
                y++;
            }
        }
        if (!isEven && coeffsLeft > 0) {
            // Handle odd row residual
            coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)] = matrices[offsetMatrix + x + y * dimsize];
            coeffsLeft--;
        }

        // Increment coordinates
        if (x_n == dimsize - 1) {
            y_n++;
        } else {
            x_n++;
        }
    }
}