#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void implantCoeffs(float* matrices, float *coeffArray, int savedCoeffs, int dimsize) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Optimize thread ID calculation to 1D

    int offsetMatrix = id * dimsize * dimsize;
    int offsetCoeff = id * savedCoeffs;
    int coeffsLeft = savedCoeffs;
    int x = 0, y = 0, y_n = 0, x_n = 1;
    int numberinrow, tmp;

    matrices[offsetMatrix] = coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)];
    coeffsLeft--;

    while (coeffsLeft > 0) {
        x = x_n;
        y = y_n;
        numberinrow = (x_n < dimsize - 1) ? (x_n + 1) : (x_n - (y_n - 1));

        while (numberinrow > 0 && coeffsLeft > 0) {
            matrices[offsetMatrix + x + y * dimsize] = coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)];
            numberinrow--;
            coeffsLeft--;

            if (numberinrow % 2 == 0) {
                tmp = x;
                x = y;
                y = tmp;
                x--;
                y++;
            } else {
                tmp = x;
                x = y;
                y = tmp;
            }
        }
        if (coeffsLeft > 0 && numberinrow == 1) {
            matrices[offsetMatrix + x + y * dimsize] = coeffArray[offsetCoeff + (savedCoeffs - coeffsLeft)];
            coeffsLeft--;
        }

        if (x_n == dimsize - 1) {
            y_n++;
        } else {
            x_n++;
        }
    }
}