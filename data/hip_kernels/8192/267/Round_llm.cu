#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Round(float * A, float *out, int size) {
    int id = blockDim.x * gridDim.x * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;

    if (id < size) {
        // Use built-in rintf function for rounding
        out[id] = rintf(out[id]);
    }
}