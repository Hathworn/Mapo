#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Prepare_1_MeansForJoin(float* input, int c_src1, int c_src2, int c_n, float* delta, int imageWidth, int imageHeight)
{
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    int size = imageWidth * imageHeight;

    // Calculate linear index if within bounds
    if (id < size) {
        int px = id % imageWidth;
        int py = id / imageWidth;

        bool insideSrc1 = delta[c_src1 * NUM_SUMS * size + 4 * size + id] != 0;
        bool insideSrc2 = delta[c_src2 * NUM_SUMS * size + 4 * size + id] != 0;

        // Simplify conditional assignments with ternary operator
        float2 pixPos = { 2.0f * px / imageWidth - 1, 2.0f * py / imageHeight - 1 };
        float inputVal = input[id];

        // Pre-calculate indices and input check
        bool validInput = inputVal > 0 && (insideSrc1 || insideSrc2);
        int baseIndex = c_n * NUM_SUMS * size + id;

        // Use conditional execution for all updates 
        delta[baseIndex + 0 * size] = validInput ? inputVal * pixPos.x : 0;
        delta[baseIndex + 1 * size] = validInput ? inputVal * pixPos.y : 0;
        delta[baseIndex + 2 * size] = validInput ? inputVal * pixPos.x * pixPos.x : 0;
        delta[baseIndex + 3 * size] = validInput ? inputVal * pixPos.y * pixPos.y : 0;
        delta[baseIndex + 4 * size] = validInput ? inputVal : 0;
    }
}