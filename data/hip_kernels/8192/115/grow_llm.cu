#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void grow(float *matrices, const int dimension, const int coefficients, const int population, float *chromosomes, const float *noise, const float mutationRate, const int kept, const float* fitnesses, int *mark, const int alpha){

    // Initialize thread ID and random state
    int threadGlobalID = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
        + threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
    int chromOffset = threadGlobalID * coefficients;
    hiprandState st;
    hiprand_init((int)noise[threadGlobalID] << threadGlobalID, threadGlobalID * (threadGlobalID == population - 1 ? noise[0] : noise[threadGlobalID]), 0, &st);
  
    if (threadGlobalID > kept - 1) {
        int parent1 = floor(hiprand_uniform(&st) * kept);
        int parent2 = floor(hiprand_uniform(&st) * kept);
        int point = floor(hiprand_uniform(&st) * coefficients);

        // Copy chromosome segments from parents
        for (int i = 0; i < point; i++) {
            chromosomes[chromOffset + i] = chromosomes[parent1 * coefficients + i];
        }
        for (int i = point; i < coefficients; i++) {
            chromosomes[chromOffset + i] = chromosomes[parent2 * coefficients + i];
        }

        // Mutate children chromosomes
        for (int i = 0; i < coefficients; i++) {
            if (hiprand_uniform(&st) <= mutationRate) {
                chromosomes[chromOffset + i] = (hiprand_uniform(&st) - 0.5f) * 2 * alpha; // Optimized mutation
            }
        }
    }

    if (threadGlobalID < coefficients) {
        // Evaluate fitness and mark genes for permutation
        float invDist = 1.0f / (fitnesses[0] - fitnesses[population - 1]);
        for (int i = 0; i < population; i++) {
            float probability = 1.0f - sqrtf((fitnesses[i] - fitnesses[population - 1]) * invDist); // Cache optimization
            mark[coefficients * i + threadGlobalID] = (hiprand_uniform(&st) < probability);
        }
        
        // Perform gene permutation
        wloc = -1;
        float tmp1 = 0.0f;
        for (int i = 0; i < population; i++) {
            if (mark[coefficients * i + threadGlobalID] == 1) {
                if (wloc == -1) {
                    wloc = i;
                    tmp1 = chromosomes[coefficients * i + threadGlobalID];
                } else {
                    float tmp2 = chromosomes[coefficients * i + threadGlobalID];
                    chromosomes[coefficients * i + threadGlobalID] = tmp1;
                    tmp1 = tmp2;
                }
            }
        }
        if (wloc != -1) {
            chromosomes[coefficients * wloc + threadGlobalID] = tmp1;
        }
    }

    __syncthreads();

    // Initialize matrices to zero
    for (int i = 0; i < dimension * dimension; i++) {
        matrices[threadGlobalID * dimension * dimension + i] = 0.0f;
    }
}