#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SetForcesToZeroKernel(float *force, int maxCells)
{
    // Calculate global thread index more efficiently
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (threadId < maxCells * 3)
    {
        force[threadId] = 0.00f;
    }
}