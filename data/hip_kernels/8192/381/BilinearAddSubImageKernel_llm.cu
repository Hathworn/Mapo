#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BilinearAddSubImageKernel(float *input, float *opImage, float* subImageDefs, int inputWidth, int inputHeight, int opImageWidth, int opImageHeight)
{
    // Calculate unique thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Load sub-image parameters
    float subImgCX = subImageDefs[0];
    float subImgCY = subImageDefs[1];
    float subImgDiameter = subImageDefs[2];

    // Calculate sub-image properties
    int maxDiameter = min(inputWidth, inputHeight);
    int diameterPix = max(1, (int)(subImgDiameter * maxDiameter));
    int subImgX = (int)(inputWidth * (subImgCX + 1) * 0.5f) - diameterPix / 2;
    int subImgY = (int)(inputHeight * (subImgCY + 1) * 0.5f) - diameterPix / 2;

    // Calculate pixel position in the sub-image
    int px = id % diameterPix;
    int py = id / diameterPix;

    // Validate pixel coordinates within the input image bounds
    if (px + subImgX >= 0 && py + subImgY >= 0 &&
        px + subImgX < inputWidth && py + subImgY < inputHeight &&
        py < diameterPix)
    {
        // Bilinear interpolation ratios
        float xRatio = (float)(opImageWidth - 1) / diameterPix;
        float yRatio = (float)(opImageHeight - 1) / diameterPix;

        int x = (int)(xRatio * px);
        int y = (int)(yRatio * py);

        float xDist = (xRatio * px) - x;
        float yDist = (yRatio * py) - y;

        // Bilinear interpolation
        float topLeft = opImage[y * opImageWidth + x];
        float topRight = opImage[y * opImageWidth + x + 1];
        float bottomLeft = opImage[(y + 1) * opImageWidth + x];
        float bottomRight = opImage[(y + 1) * opImageWidth + x + 1];

        float result = 
            topLeft * (1 - xDist) * (1 - yDist) +
            topRight * xDist * (1 - yDist) +
            bottomLeft * yDist * (1 - xDist) +
            bottomRight * xDist * yDist;

        // Add result to input
        atomicAdd(&input[(py + subImgY) * inputWidth + px + subImgX], result); // Use atomicAdd to prevent race conditions
    }
}