#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FloorKernel(float* input, float* output, int size)
{
    // Calculate global thread index using 1D grid
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread index is within bounds
    if (id < size)
    {
        output[id] = floorf(input[id]); // Use floorf for performance
    }
}