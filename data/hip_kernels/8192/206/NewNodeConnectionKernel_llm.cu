#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NewNodeConnectionKernel(int f, int q, int r, int *activityFlag, int *connection, int *age, float *localError, float alfa, int maxCells, float errorFraction)
{
    // Calculate the global thread ID
    int threadId = blockDim.x * blockIdx.y * gridDim.x  // rows preceding current row in grid
                 + blockDim.x * blockIdx.x              // blocks preceding current block
                 + threadIdx.x;

    // Only process the first thread
    if (threadId == 0)
    {
        // Set activity flag
        activityFlag[r] = 1;

        // Reset and update connections and ages
        int indices[] = {q * maxCells + f, f * maxCells + q, q * maxCells + r, r * maxCells + q, f * maxCells + r, r * maxCells + f};
        for (int i = 0; i < 6; ++i)
        {
            connection[indices[i]] = (i >= 2);  // set connection for (q, r), (r, q), (f, r), (r, f) to 1
            age[indices[i]] = 0;  // reset ages
        }

        // Update local errors
        localError[q] *= (1 - alfa);
        localError[f] *= (1 - alfa);
        localError[r] = errorFraction * (localError[q] + localError[f]);
    }
}