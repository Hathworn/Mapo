#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CeilKernel(float* input, float* output, int size)
{
    // Calculate id using a more efficient computation
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockDim.x * blockDim.y * blockIdx.y;

    // Check and compute if within bounds
    if(id < size)
    {
        output[id] = ceilf(input[id]); // Use ceilf for single precision
    }
}