#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LinearCombinationKernel(float *input1, float input1_coeff, int input1_start_index, float *input2, float input2_coeff, int input2_start_index, float *output, int output_start_index, int size)
{
    // Calculate the global thread index using a flattened 1D grid
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we are operating within the size bound
    if (id < size)
    {
        // Perform the linear combination
        output[output_start_index + id] = input1_coeff * input1[input1_start_index + id] + input2_coeff * input2[input2_start_index + id];
    }
}