#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ExactResampleKernel_1toN(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
    // Calculate linear index based on block and thread indices
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    int size = outputWidth * outputHeight;

    if (id < size)
    {
        // Calculate output point coordinates
        int px = id % outputWidth;
        int py = id / outputWidth;

        // Precompute ratios
        float xRatio = static_cast<float>(inputWidth) / outputWidth;
        float yRatio = static_cast<float>(inputHeight) / outputHeight;

        // Compute corresponding coordinates in the original image
        int x = min(static_cast<int>(px * xRatio), inputWidth - 1); // Ensure within bounds
        int y = min(static_cast<int>(py * yRatio), inputHeight - 1); // Ensure within bounds

        output[py * outputWidth + px] = input[y * inputWidth + x];
    }
}