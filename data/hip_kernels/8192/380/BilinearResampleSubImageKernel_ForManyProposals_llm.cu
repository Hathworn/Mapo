#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void BilinearResampleSubImageKernel_ForManyProposals(const float *input, float *output, const float* subImageDefs, bool safeBounds, int subImageDefsDim, int inputWidth, int inputHeight, int outputWidth, int outputHeight, int numberSubImages, int outputSize)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x; // Simplify ID calculation

    if (id < outputSize)
    {
        int subim_id = id / (outputWidth * outputHeight);
        int idx = id % (outputWidth * outputHeight);
        int py = idx / outputWidth;  
        int px = idx % outputWidth;  

        float subImgCX = subImageDefs[subim_id * subImageDefsDim];
        float subImgCY = subImageDefs[1 + subim_id * subImageDefsDim];
        float subImgDiameter = subImageDefs[2 + subim_id * subImageDefsDim];

        int maxDiameter = min(inputWidth - 1, inputHeight - 1);
        int diameterPix = max(1, min(maxDiameter, (int)(subImgDiameter * maxDiameter)));

        int subImgX = (int)(inputWidth * (subImgCX + 1) * 0.5f) - diameterPix / 2;
        int subImgY = (int)(inputHeight * (subImgCY + 1) * 0.5f) - diameterPix / 2;

        if (safeBounds)
        {
            subImgX = max(subImgX, 1);
            subImgY = max(subImgY, 1);
            subImgX = min(subImgX, inputWidth - diameterPix - 1);
            subImgY = min(subImgY, inputHeight - diameterPix - 1);
        }

        float xRatio = (float)(diameterPix - 1) / (outputWidth - 1);
        float yRatio = (float)(diameterPix - 1) / (outputHeight - 1);

        float xDist, yDist;
        int x = (int)(xRatio * px);
        int y = (int)(yRatio * py);
        
        if ((x + subImgX) >= 0 && (y + subImgY) >= 0 && (x + subImgX) < inputWidth && (y + subImgY) < inputHeight)
        {
            xDist = (xRatio * px) - x;
            yDist = (yRatio * py) - y;

            float topLeft = input[(y + subImgY) * inputWidth + x + subImgX];
            float topRight = input[(y + subImgY) * inputWidth + x + subImgX + 1];
            float bottomLeft = input[(y + subImgY + 1) * inputWidth + x + subImgX];
            float bottomRight = input[(y + subImgY + 1) * inputWidth + x + subImgX + 1];

            float result = topLeft * (1 - xDist) * (1 - yDist) +
                           topRight * xDist * (1 - yDist) +
                           bottomLeft * yDist * (1 - xDist) +
                           bottomRight * xDist * yDist;

            output[py * outputWidth + px + subim_id * outputWidth * outputHeight] = result;
        }
    }
}