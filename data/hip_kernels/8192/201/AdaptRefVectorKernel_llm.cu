#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AdaptRefVectorKernel(int cell, float *referenceVector, float oldErrorFraction, float youngErrorFraction, float decayFactor, int *winningCount, float *difference, int inputSize)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the unique thread ID

    if (threadId < inputSize)
    {
        // Precompute exponential decay factor outside the loop
        float decayedError = expf(-decayFactor * winningCount[cell]);
        float errorFraction = (youngErrorFraction - oldErrorFraction) * decayedError + oldErrorFraction;

        // Use pointer arithmetic for accessing elements
        float *refVecElement = referenceVector + cell * inputSize + threadId;
        float *diffElement = difference + cell * inputSize + threadId;

        // Update the reference vector
        *refVecElement += errorFraction * (*diffElement);
    }
}