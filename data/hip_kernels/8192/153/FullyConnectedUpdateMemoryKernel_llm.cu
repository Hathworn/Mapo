#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FullyConnectedUpdateMemoryKernel( float *avgWeightGradPtr, float *avgBiasGradPtr, float *avgWeightGradVarPtr, float *avgBiasGradVarPtr, float *weightMemorySizePtr, float *biasMemorySizePtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize )
{
    int j = blockDim.x * blockIdx.x + threadIdx.x; // Calculate global thread index

    if (j < thisLayerSize && !dropoutMaskPtr[j]) // Check thread range and mask in a single condition
    {
        float avgBiasGrad = avgBiasGradPtr[j];
        float avgBiasGradVar = avgBiasGradVarPtr[j];
        float biasMemory = biasMemorySizePtr[j];

        for (int i = 0; i < prevLayerSize; i++)
        {
            int index = i * thisLayerSize + j; // Compute index once per inner loop

            // Cache reused values to avoid redundant memory accesses
            float avgWeightGrad = avgWeightGradPtr[index];
            float avgWeightGradVar = avgWeightGradVarPtr[index];

            // Update memory size with cached values
            weightMemorySizePtr[index] = (1.0f - avgWeightGrad * avgWeightGrad / avgWeightGradVar) * weightMemorySizePtr[index] + 1.0f;
        }

        // Update bias memory size with cached values
        biasMemorySizePtr[j] = (1.0f - avgBiasGrad * avgBiasGrad / avgBiasGradVar) * biasMemory + 1.0f;
    }
}