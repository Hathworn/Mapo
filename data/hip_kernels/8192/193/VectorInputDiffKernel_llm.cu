#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VectorInputDiffKernel( float *input, int inputSize, float *referenceVector, int maxCells, float *difference )
{
    // Calculate global thread index
    int threadId = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Ensure threadId is within valid range
    if (threadId < maxCells * inputSize)
    {
        // Compute the difference by leveraging modular arithmetic.
        int inputIdx = threadId % inputSize; // Compute index for input
        difference[threadId] = input[inputIdx] - referenceVector[threadId];
    }
}