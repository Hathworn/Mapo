#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FullyConnectedShiftKernel(float *weightPtr, float *biasPtr, float *shiftedWeightsPtr, float *shiftedBiasPtr, float *avgWeightGradPtr, float *avgBiasGradPtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize)
{
    int j = blockDim.x * blockIdx.y * gridDim.x  // Calculate global thread index
            + blockDim.x * blockIdx.x 
            + threadIdx.x;

    if (j < thisLayerSize && !dropoutMaskPtr[j]) // Check boundary and dropout condition
    {
        int index = j;
        for (int i = 0; i < prevLayerSize; i++) // Optimize loop by removing i declaration outside
        {
            shiftedWeightsPtr[index] = weightPtr[index] + avgWeightGradPtr[index]; // Update weight gradient
            index += thisLayerSize;
        }

        shiftedBiasPtr[j] = biasPtr[j] - avgBiasGradPtr[j]; // Update bias gradient
    }
}