#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RBMCopyFilterKernel(float *weightPtr, float *filterPtr, int weightCount, int i, int thisLayerSize)
{
    // Calculate the linear thread index in grid
    int weightIndex = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Ensure computation only within bounds
    if (weightIndex < weightCount)
    {
        filterPtr[weightIndex] = weightPtr[i + weightIndex * thisLayerSize];
    }
}