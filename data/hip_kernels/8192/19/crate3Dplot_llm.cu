#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void crate3Dplot(float* plotValues, float patchSize, int itemsX, int itemsY, float maxValue, float* vertexData)
{
    // Calculate threadId and total size
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    int size = itemsX * itemsY;

    // Early exit if threadId is out of bounds
    if (threadId >= size) return;

    // Precompute some constants and indices
    float texSizeX = 1.0f / itemsX;
    float texSizeY = 1.0f / itemsY;
    int patchX = threadId / itemsY;
    int patchY = itemsY - (threadId % itemsY) - 1;
    float height = plotValues[threadId] / maxValue;
    float3* vertTop = (float3*)vertexData;
    float2* texCoords = (float2*)(vertexData + (60 * size));

    // Shared gap variable
    const float gap = 0.000;

    // Calculate positions for top side vertices
    float3 topVertices[4] = {
        {patchX * patchSize + gap, height, patchY * patchSize + gap},
        {(patchX + 1) * patchSize - gap, height, patchY * patchSize + gap},
        {(patchX + 1) * patchSize - gap, height, (patchY + 1) * patchSize - gap},
        {patchX * patchSize + gap, height, (patchY + 1) * patchSize - gap}
    };

    // Assign top side vertices and texture coordinates
    for (int i = 0; i < 4; ++i) {
        vertTop[threadId * 4 + i] = topVertices[i];
    }

    texCoords[threadId * 4] = {patchX * texSizeX, patchY * texSizeY};
    texCoords[threadId * 4 + 1] = {(patchX + 1) * texSizeX, patchY * texSizeY};
    texCoords[threadId * 4 + 2] = {(patchX + 1) * texSizeX, (patchY + 1) * texSizeY};
    texCoords[threadId * 4 + 3] = {patchX * texSizeX, (patchY + 1) * texSizeY};

    // Define other side vertices offsets
    float3* vertLeft = (float3*)(vertexData + 12 * size);
    float3* vertFar = (float3*)(vertexData + 24 * size);
    float3* vertNear = (float3*)(vertexData + 36 * size);
    float3* vertRight = (float3*)(vertexData + 48 * size);

    float3 bottomVertex, sideVertices[2];

    // Initialize the sides
    for (int i = 0; i < 5; ++i) {
        switch (i) {
            // Left side
            case 0:
                sideVertices[0] = topVertices[0];
                sideVertices[1] = topVertices[3];
                bottomVertex = topVertices[0];
            break;
            // Far side
            case 1:
                sideVertices[0] = topVertices[2];
                sideVertices[1] = topVertices[3];
                bottomVertex = topVertices[2];
            break;
            // Near side
            case 2:
                sideVertices[0] = topVertices[1];
                sideVertices[1] = topVertices[0];
                bottomVertex = topVertices[1];
            break;
            // Right side
            case 3:
                sideVertices[0] = topVertices[2];
                sideVertices[1] = topVertices[1];
                bottomVertex = topVertices[2];
            break;
        }
        
        sideVertices[0].y = sideVertices[1].y = bottomVertex.y = 0;

        // Assign vertices for each side
        switch (i) {
            case 0: {
                float3* vertices = vertLeft + threadId * 4;
                vertices[0] = bottomVertex;
                vertices[1] = topVertices[0];
                vertices[2] = sideVertices[1];
                vertices[3] = topVertices[3];
            } break;
            case 1: {
                float3* vertices = vertFar + threadId * 4;
                vertices[0] = bottomVertex;
                vertices[1] = sideVertices[1];
                vertices[2] = topVertices[3];
                vertices[3] = topVertices[2];
            } break;
            case 2: {
                float3* vertices = vertNear + threadId * 4;
                vertices[0] = bottomVertex;
                vertices[1] = sideVertices[1];
                vertices[2] = topVertices[0];
                vertices[3] = topVertices[1];
            } break;
            case 3: {
                float3* vertices = vertRight + threadId * 4;
                vertices[0] = bottomVertex;
                vertices[1] = topVertices[2];
                vertices[2] = sideVertices[1];
                vertices[3] = topVertices[1];
            } break;
        }
    }
}