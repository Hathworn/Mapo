#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getRow_optimized(const float * A, float * row_id, float * out, int Acols) {
    // Calculate the unique global thread ID
    int id = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x;
    
    // Ensure that the thread accesses within bounds
    if (id < Acols) {
        // Compute the base index once outside the loop to enhance performance
        int baseIndex = static_cast<int>(*row_id) * Acols;
        out[id] = A[id + baseIndex];
    }
}