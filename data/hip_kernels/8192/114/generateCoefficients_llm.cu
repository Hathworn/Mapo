#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generateCoefficients(float *chromosomes, const int chromSize, const float* noise, const int population, const int alpha){

    // Calculate global thread ID
    int threadGlobalID = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
                         + threadIdx.z * blockDim.y * blockDim.x 
                         + threadIdx.y * blockDim.x 
                         + threadIdx.x;

    // Early exit if thread ID is out of population bounds
    if (threadGlobalID >= population) return;

    // Initialize random state with more coherence
    hiprandState st;
    hiprand_init((int)noise[threadGlobalID] ^ threadGlobalID, threadGlobalID, 0, &st);

    // Generate coefficients only if thread ID is valid
    for (int i = 0; i < chromSize; i++){
        float randomValue = hiprand_uniform(&st) * alpha;
        chromosomes[chromSize * threadGlobalID + i] = (hiprand_uniform(&st) < 0.5) ? randomValue : -randomValue;
    }
}