#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CopyInputToVisFieldKernel(float *input, float *visField, int inputSize)
{
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x; // Simplified thread ID calculation

    if (threadId < inputSize) 
    {
        visField[threadId] = input[threadId]; // Directly copy input to visField
    }
}