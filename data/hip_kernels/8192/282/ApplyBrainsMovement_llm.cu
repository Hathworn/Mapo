#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ApplyBrainsMovement(float *CCXY, int dim_XY, float *movement, int dim_movement, int max_clusters) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Compute 1D global thread ID
    if (id < max_clusters) { // Check for valid cluster index within bounds
        if (dim_movement >= 2) { // Check for movement in XY
            float m0 = movement[0]; // Load movement values into registers for faster access
            float m1 = movement[1];
            CCXY[id * dim_XY] -= m0; // Apply movement in X
            CCXY[id * dim_XY + 1] -= m1; // Apply movement in Y
        }
        if (dim_movement >= 3) { // Placeholder for additional dimensions, if needed
            // Future work: Apply additional transformations in X
        }
    }
}