#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Matrix_PermuteRows(const float * A, int Acount, int Acols, const float * B, int Bcount, int Bcols, float * out0, int out0count, int out0cols)
{
    // Optimization: Calculate global thread ID using efficient 2D grid indexing.
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    
    if (id < Acount)
    {
        int id_row = id / Acols;
        int id_col = id % Acols;
        
        // Optimization: Precompute new row index once per thread and avoid repeating multiplications.
        int id_rowNew = __ldg(&B[id_row]) * Acols;
        
        // Optimization: Use __ldg for potentially faster global memory loads.
        out0[id] = __ldg(&A[id_col + id_rowNew]);
    }
}