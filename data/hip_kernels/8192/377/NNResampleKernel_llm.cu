#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NNResampleKernel(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
    // Calculate pixel index
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int size = outputWidth * outputHeight;

    // Ensure we are within bounds
    if (id < size)
    {
        int px = id % outputWidth;
        int py = id / outputWidth;

        // Precompute ratios for efficiency
        float xRatio = (float)(inputWidth - 1) / outputWidth;
        float yRatio = (float)(inputHeight - 1) / outputHeight;

        // Calculate nearest neighbor indices
        int x = (int)(xRatio * px + 0.5f);
        int y = (int)(yRatio * py + 0.5f);

        // Remap the pixel
        output[id] = input[y * inputWidth + x];
    }
}