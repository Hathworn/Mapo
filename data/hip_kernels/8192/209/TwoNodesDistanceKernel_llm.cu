#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TwoNodesDistanceKernel(float *twoNodesDifference, float *twoNodesDistance, int vectorLength) {
    int threadId = blockDim.x * blockIdx.y * gridDim.x // rows preceeding current row in grid
                  + blockDim.x * blockIdx.x           // blocks preceeding current block
                  + threadIdx.x;

    if (threadId < 1) {
        float sum = 0.00f;
        // Loop unrolling for better performance
        int i = 0;
        for (; i <= vectorLength - 4; i += 4) {
            float v1 = twoNodesDifference[threadId * vectorLength + i];
            float v2 = twoNodesDifference[threadId * vectorLength + i + 1];
            float v3 = twoNodesDifference[threadId * vectorLength + i + 2];
            float v4 = twoNodesDifference[threadId * vectorLength + i + 3];
            sum += v1 * v1 + v2 * v2 + v3 * v3 + v4 * v4;
        }
        for (; i < vectorLength; i++) {
            float value = twoNodesDifference[threadId * vectorLength + i];
            sum += value * value;
        }
        twoNodesDistance[threadId] = sqrtf(sum);
    }
}