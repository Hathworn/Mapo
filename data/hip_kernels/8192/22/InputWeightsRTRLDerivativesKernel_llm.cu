#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InputWeightsRTRLDerivativesKernel(float *input, float *hiddenActivationDerivatives, float *recurrentWeights, float *inputWeightRTRLDerivatives, float *previousInputWeightRTRLDerivatives)
{
    int partialId = blockDim.x * blockIdx.y * gridDim.x  // Calculate unique thread ID
                  + blockDim.x * blockIdx.x
                  + threadIdx.x;

    if (partialId < D_HIDDEN_UNITS * D_HIDDEN_UNITS * D_INPUT_UNITS)
    {
        int unitId = partialId / (D_HIDDEN_UNITS * D_INPUT_UNITS);
        int weightId = partialId % (D_HIDDEN_UNITS * D_INPUT_UNITS);
        int to = weightId / D_INPUT_UNITS;
        int from = weightId % D_INPUT_UNITS;

        // Optimized memory access by avoiding bank conflicts and using local registers
        float sum = 0;
        for (int i = 0; i < D_HIDDEN_UNITS; i++)
        {
            // Load values into local registers
            float recWeight = recurrentWeights[unitId * D_HIDDEN_UNITS + i];
            float prevInputWeight = previousInputWeightRTRLDerivatives[i * (D_HIDDEN_UNITS * D_INPUT_UNITS) + weightId];
            sum += recWeight * prevInputWeight;
        }

        // Calculate the RTRL derivative
        inputWeightRTRLDerivatives[partialId] = hiddenActivationDerivatives[unitId] * ((unitId == to) * input[from] + sum);
    }
}