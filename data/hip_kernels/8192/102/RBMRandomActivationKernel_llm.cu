#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float activateRandomly(float probability, float random)
{
    return random < probability;
}

__global__ void RBMRandomActivationKernel(float *outputPtr, float *randomPtr, int size)
{
    // Calculate flattened index using grid-stride loop
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; 

    for (; i < size; i += stride)
    {
        outputPtr[i] = activateRandomly(outputPtr[i], randomPtr[i]);
    }
}