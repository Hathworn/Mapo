#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SetAllButOneKernel(float *buffer, float value, int index, int count)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Optimize threadId calculation

    if (threadId < count && threadId != index)
    {
        buffer[threadId] = value;
    }
}