#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_getRow_FloatPointer_optimized(const float * A, int Acount, int Acols, const float * rowId, int empty_par1, int empty_par2, float * out0, int out0count, int out0cols)
{
    // Calculate unique thread id with fewer operations
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread id is within bounds
    if (id < Acols)
    {
        // Optimize memory access pattern by reducing unnecessary operations
        int rowIndex = static_cast<int>(rowId[0]);
        out0[id] = A[id + rowIndex * Acols];
    }
}