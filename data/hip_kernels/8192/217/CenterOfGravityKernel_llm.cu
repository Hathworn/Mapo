#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CenterOfGravityKernel(float *pointsCoordinates, float *centerOfGravity, int *activityFlag, int maxCells)
{
    int threadId = blockDim.x * blockIdx.y * gridDim.x  // calculate global thread ID
                 + blockDim.x * blockIdx.x
                 + threadIdx.x;

    // Use a single thread to compute the center of gravity
    if (threadId == 0) 
    {
        float xSum = 0.0f, ySum = 0.0f, zSum = 0.0f;
        int livingCells = 0;
        
        // Unrolling loop to optimize
        for (int c = 0; c < maxCells - 1; c += 2)
        {
            if (activityFlag[c] == 1) {
                xSum += pointsCoordinates[c * 3];
                ySum += pointsCoordinates[c * 3 + 1];
                zSum += pointsCoordinates[c * 3 + 2];
                livingCells++;
            }
            if (activityFlag[c + 1] == 1) {
                xSum += pointsCoordinates[(c + 1) * 3];
                ySum += pointsCoordinates[(c + 1) * 3 + 1];
                zSum += pointsCoordinates[(c + 1) * 3 + 2];
                livingCells++;
            }
        }

        // Handle the last element if maxCells is odd
        if (maxCells % 2 != 0) {
            int c = maxCells - 1;
            if (activityFlag[c] == 1) {
                xSum += pointsCoordinates[c * 3];
                ySum += pointsCoordinates[c * 3 + 1];
                zSum += pointsCoordinates[c * 3 + 2];
                livingCells++;
            }
        }

        if (livingCells > 0) {  // Ensure no division by zero
            centerOfGravity[0] = xSum / (float)livingCells;
            centerOfGravity[1] = ySum / (float)livingCells;
            centerOfGravity[2] = zSum / (float)livingCells;
        }
    }
}