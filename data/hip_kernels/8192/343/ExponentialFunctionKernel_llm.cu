#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ExponentialFunctionKernel(float exponent, float* input, float* output, int size)
{
    // Use single dimensional grid for easier threading
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid recomputing the block index for each dimension
    if (id < size)
    {
        // Use expf for single precision exponential calculation
        output[id] = expf(exponent * logf(input[id]));
    }
}