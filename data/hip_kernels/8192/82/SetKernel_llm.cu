#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SetKernel(float *buffer, int offset, float value, int count)
{
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;  // Calculate linear thread ID

    if (threadId < count)  // Ensure thread ID is within bounds
    {
        buffer[threadId + offset] = value;  // Write value to the corresponding buffer position
    }
}