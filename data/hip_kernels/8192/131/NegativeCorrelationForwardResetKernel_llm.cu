#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NegativeCorrelationForwardResetKernel(float* outputPtr, int thisLayerSize)
{
    // Calculate the global thread index
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds and reset output
    if (j < thisLayerSize)
    {
        outputPtr[j] = 0;
    }
}