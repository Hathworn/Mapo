#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DrawRgbaColorKernel(float *target, int targetWidth, int targetHeight, int inputX, int inputY, int areaWidth, int areaHeight, float r, float g, float b) {
    // Calculate global thread ID
    int id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    // Pre-compute useful values
    int targetPixels = targetWidth * targetHeight;
    int texturePixels = areaWidth * areaHeight;

    // Determine RGB channel and texture position
    int idTextureRgb = id / texturePixels;
    if (idTextureRgb < 3) {
        int idTexturePixel = id % texturePixels;
        int idTextureY = idTexturePixel / areaWidth;
        int idTextureX = idTexturePixel % areaWidth;

        // Boundary check for valid position in target
        int targetX = idTextureX + inputX;
        int targetY = idTextureY + inputY;

        if (targetX < targetWidth && targetX >= 0 && targetY < targetHeight && targetY >= 0) {
            // Select color based on channel
            float color = (idTextureRgb == 0) ? r : (idTextureRgb == 1) ? g : b;

            // Calculate target index and assign color
            int tIndex = targetPixels * idTextureRgb + targetWidth * targetY + targetX;
            target[tIndex] = color;
        }
    }
}