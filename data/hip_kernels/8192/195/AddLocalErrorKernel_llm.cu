#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AddLocalErrorKernel( int s1, float *distance, float *localError )
{
    // Calculate the global thread index
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Only one thread contributes to updating localError[s1]
    if (threadId == 0)
    {
        atomicAdd(&localError[s1], distance[s1] * distance[s1]);  // Use atomicAdd for safe concurrent updates
    }
}