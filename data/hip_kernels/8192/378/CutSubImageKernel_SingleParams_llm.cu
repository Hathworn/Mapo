#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CutSubImageKernel_SingleParams(float *input, float *output, float subImageX, float subImageY, float subImageDiameter, bool safeBounds, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
    int id = blockDim.x * blockIdx.y * gridDim.x + blockDim.x * blockIdx.x + threadIdx.x;
    int size = outputWidth * outputHeight;

    if (id < size)
    {
        // Pre-calculate constants that do not change
        float subImgCX = subImageX; // <-1, 1>
        float subImgCY = subImageY; // <-1, 1>
        int maxDiameter = min(inputWidth - 1, inputHeight - 1);
        int diameterPix = max(1, min((int)(subImageDiameter * maxDiameter), maxDiameter));
        int subImgX = (int)(inputWidth * (subImgCX + 1) * 0.5f) - diameterPix / 2;
        int subImgY = (int)(inputHeight * (subImgCY + 1) * 0.5f) - diameterPix / 2;

        if (safeBounds)
        {
            subImgX = max(subImgX, 1);
            subImgY = max(subImgY, 1);
            subImgX = min(subImgX, inputWidth - diameterPix - 1);
            subImgY = min(subImgY, inputHeight - diameterPix - 1);
        }

        int px = id % outputWidth;
        int py = id / outputWidth;
        // Use float division only once
        float xRatio = ((float)(diameterPix - 1)) / (outputWidth - 1);
        float yRatio = ((float)(diameterPix - 1)) / (outputHeight - 1);

        int x = (int)(xRatio * px);
        int y = (int)(yRatio * py);

        // Check bounds before accessing the input
        if (x + subImgX >= 0 && y + subImgY >= 0 && x + subImgX < inputWidth && y + subImgY < inputHeight)
        {
            output[py * outputWidth + px] = input[(y + subImgY) * inputWidth + x + subImgX];
        }
    }
}