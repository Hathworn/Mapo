#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Multiply(float* im, float val, int size)
{
    // Calculate a unique global index for each thread
    int id = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;

    // Perform computation only if id is within bounds
    if (id < size)
    {
        // Multiply image value by the given scalar
        im[id] *= val;
    }
}