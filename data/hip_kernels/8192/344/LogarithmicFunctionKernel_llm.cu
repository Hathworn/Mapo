#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LogarithmicFunctionKernel(float* input, float* output, int size, const int type)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate thread index more efficiently

    if (id < size)
    {
        float val = input[id];  // Reduce repeated memory access
        switch (type)
        {
            case 1:
                output[id] = logf(val);
                break;
            case 2:
                output[id] = log2f(val);
                break;
            case 3:
                output[id] = log10f(val);
                break;
        }
    }
}