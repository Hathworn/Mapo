#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeBiasTermKernel(float *biasTerm, float cFactor, float *winningFraction, int activeCells, int maxCells)
{
    // Use a single dimensional block and grid structure for simplicity
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadId < maxCells)
    {
        // Precompute common expression outside the loop for efficiency
        float factor = cFactor / activeCells;
        biasTerm[threadId] = factor - cFactor * winningFraction[threadId];
    }
}