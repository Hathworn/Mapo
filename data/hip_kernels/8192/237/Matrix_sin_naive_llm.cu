#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_sin_optimized(const float *A, int Acount, int Acols, float *out0, int out0count)
{
    // Compute id using a continuous, linear grid-stride loop for higher occupancy
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = id; i < out0count; i += stride) {
        out0[i] = sin(A[i]);
    }
}