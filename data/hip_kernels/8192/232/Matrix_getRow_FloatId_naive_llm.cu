#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_getRow_FloatId_optimized(const float * A, int Acount, int Acols, float * out0, int out0count, int out0cols, const float row_id)
{
    // Calculate global thread ID effectively using 2D block and grid
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if within bounds
    if (id < Acols) 
    {
        // Compute row offset once and reuse
        int rowOffset = __fmaf_rd(row_id, Acols, id); // FMA for address calculation
        // Load matrix element and set output
        out0[id] = A[rowOffset];
    }
}