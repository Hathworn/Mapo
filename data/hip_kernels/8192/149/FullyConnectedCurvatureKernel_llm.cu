#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FullyConnectedCurvatureKernel( float *weightsGradPtr, float *biasGradPtr, float *shiftedWeightsPtr, float *shiftedBiasPtr, float *avgWeightGradPtr, float *avgBiasGradPtr, float *weightGradCurvePtr, float *biasGradCurvePtr, float *dropoutMaskPtr, int prevLayerSize, int thisLayerSize )
{
    // Precompute common indices and load once
    int j = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    if (j < thisLayerSize)
    {
        if (!dropoutMaskPtr[j])
        {
            int index = j;
            float avgGradWeights, avgGradBias;

            for (int i = 0; i < prevLayerSize; i++)
            {
                // weight finite difference curvature
                avgGradWeights = avgWeightGradPtr[index];
                avgGradWeights = (avgGradWeights == 0) ? 0.000001f : avgGradWeights; // safe division
                weightGradCurvePtr[index] = fabsf(weightsGradPtr[index] - shiftedWeightsPtr[index]) / avgGradWeights;
                index += thisLayerSize;
            }

            // bias finite difference curvature
            avgGradBias = avgBiasGradPtr[j];
            avgGradBias = (avgGradBias == 0) ? 0.000001f : avgGradBias; // safe division
            biasGradCurvePtr[j] = fabsf(biasGradPtr[j] - shiftedBiasPtr[j]) / avgGradBias;
        }
    }
}
```
