#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DrawMaskedColorKernel2DBlock(float *target, int targetWidth, int targetHeight, int inputX, int inputY, float *textureMask, int textureWidth, int textureHeight, float r, float g, float b)
{
    // Calculate global thread indices for 2D blocks
    int idTextureX = blockIdx.x * blockDim.x + threadIdx.x;
    int idTextureY = blockIdx.y * blockDim.y + threadIdx.y;

    // Only process pixels within the texture dimensions
    if (idTextureX < textureWidth && idTextureY < textureHeight) 
    {
        // Compute texture RGB channel index and target index offset
        int idTextureRgb = blockIdx.z;
        int texturePixels = textureWidth * textureHeight;
        int targetPixels = targetWidth * targetHeight;
        int tIndex = targetPixels * idTextureRgb + targetWidth * (idTextureY + inputY) + (idTextureX + inputX);
        int aIndex = idTextureY * textureWidth + idTextureX + 3 * texturePixels; // A component index

        // Check bounds and alpha mask value
        if (idTextureRgb < 3 && idTextureX + inputX < targetWidth && idTextureX + inputX >= 0 && 
            idTextureY + inputY < targetHeight && idTextureY + inputY >= 0) 
        {
            float a = textureMask[aIndex];
            if (a > 0) 
            {
                switch (idTextureRgb)
                {
                    case 0:
                        target[tIndex] = r;
                        break;
                    case 1:
                        target[tIndex] = g;
                        break;
                    case 2:
                        target[tIndex] = b;
                        break;
                }
            }
        }
    }
}