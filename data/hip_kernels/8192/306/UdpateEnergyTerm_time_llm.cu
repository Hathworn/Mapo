#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UdpateEnergyTerm_time(float* energy, int energy_dim, int nPatches, float* idFocuser_focused, float par_time_increase_energy_on_focus, float par_time_decrease_energy_in_time)
{
    // Compute global thread index directly with less computation
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    if (id < energy_dim * nPatches) {
        int idDim = id % energy_dim;
        int idPatch = id / energy_dim;
        if (idDim == 0) { // Time dimension check
            // Use ternary operator for concise conditional energy update
            energy[id] = (idPatch == static_cast<int>(*idFocuser_focused)) ? 
                         (energy[id] + par_time_increase_energy_on_focus) : 
                         (energy[id] / par_time_decrease_energy_in_time);
        }
    }
}