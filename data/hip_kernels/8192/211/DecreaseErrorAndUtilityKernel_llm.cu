#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DecreaseErrorAndUtilityKernel(float *localError, float *utility, int *activityFlag, int maxCells, float beta)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x; // Calculate thread ID

    // Ensure threadId is within bounds and active
    if(threadId < maxCells && activityFlag[threadId] == 1)
    {
        float error = localError[threadId];
        float util = utility[threadId];
        
        // Update values with reduced memory accesses
        localError[threadId] = error - beta * error;
        utility[threadId] = util - beta * util;
    }
}