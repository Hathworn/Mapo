#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Copy_matA_to_matB_withShuffleIdx(float *A, float *B, int size, int cols, float *new_idxs, int max_rows) {
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    int irow = id / cols;
    int icol = id % cols;

    // Ensure mapping only valid threads
    if (id < size) {
        int irow_new = max_rows - 1 - irow;
        int irow_old = static_cast<int>(new_idxs[irow]); // Cast float idx to int
        B[irow_new * cols + icol] = A[irow_old * cols + icol];
    }
}