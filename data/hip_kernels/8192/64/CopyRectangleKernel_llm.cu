#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CopyRectangleKernel(float *src, int srcOffset, int srcWidth, int srcRectX, int srcRectY, 
                                    int rectWidth, int rectHeight, float *dest, int destOffset, int destWidth, 
                                    int destRectX, int destRectY) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global x index
    int y = blockIdx.y * blockDim.y + threadIdx.y; // Calculate global y index

    // Optimize: Calculate linear index and ensure it is within bounds
    int id = y * rectWidth + x;
    if (x < rectWidth && y < rectHeight) {

        int srcPixelX = srcRectX + x;
        int srcPixelY = srcRectY + y;

        int destPixelX = destRectX + x;
        int destPixelY = destRectY + y;

        // Perform the copy
        dest[destOffset + destPixelX + destPixelY * destWidth] = 
            src[srcOffset + srcPixelX + srcPixelY * srcWidth];
    }
}