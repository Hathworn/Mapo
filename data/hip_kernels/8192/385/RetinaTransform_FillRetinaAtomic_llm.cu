#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void EstimateParForSubsample(float* subImageDefs, bool safeBounds, int inputWidth, int inputHeight, int2 & subImg, int & diameterPix)
{
    diameterPix = (int)(fminf((float)inputWidth, (float)inputHeight) * subImageDefs[2]); // <0,1>

    subImg.x = (int)((float)inputWidth * (subImageDefs[0] + 1) * 0.5f);
    subImg.y = (int)((float)inputHeight * (subImageDefs[1] + 1) * 0.5f);

    int maxDiameter = min(inputWidth - 1, inputHeight - 1);

    diameterPix = max(1, diameterPix);
    diameterPix = min(maxDiameter, diameterPix);

    if (safeBounds)
    {
        subImg.x = max(subImg.x, 1);
        subImg.y = max(subImg.y, 1);
        subImg.x = min(subImg.x, inputWidth - diameterPix - 1);
        subImg.y = min(subImg.y, inputHeight - diameterPix - 1);
    }
}

__global__ void RetinaTransform_FillRetinaAtomic(float* subImageDefs, float* input, int inputWidth, int inputHeight, float* output, int outputDataSize, float* retinaMask, int retinaDataSize, int retinaMaskColHint, float* retinaDataInserted)
{
    int id_pxl = blockDim.x * blockIdx.y * gridDim.x + blockDim.x * blockIdx.x + threadIdx.x;

    if (id_pxl < inputWidth * inputHeight)
    {
        int2 subImg;
        int diameterPix;
        bool safeBounds = false; // Prefer "false" for readability

        EstimateParForSubsample(subImageDefs, safeBounds, inputWidth, inputHeight, subImg, diameterPix);

        int x = id_pxl % inputWidth;
        int y = id_pxl / inputWidth;

        float minDist = FLT_MAX; // Use predefined max float for readability
        int minIdx = 0;

        #pragma unroll
        for (int id_retinaPoint = 0; id_retinaPoint < retinaDataSize; id_retinaPoint++)
        {
            float x_mask = (retinaMask[id_retinaPoint * retinaMaskColHint] * diameterPix) + subImg.x;
            float y_mask = (retinaMask[id_retinaPoint * retinaMaskColHint + 1] * diameterPix) + subImg.y;

            float dist = (x - x_mask) * (x - x_mask) + (y - y_mask) * (y - y_mask);

            if (dist < minDist)
            {
                minDist = dist;
                minIdx = id_retinaPoint;
            }
        }
        atomicAdd(&output[minIdx], input[id_pxl]);
        atomicAdd(&retinaDataInserted[minIdx], 1);
    }
}