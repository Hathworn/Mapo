#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeQuadsKernel(float *pointsCoordinates, float *vertexData, int quadOffset, float textureSide, int *activityFlag, int textureWidth, int maxCells)
{
    // Compute threadId using simpler and clearer syntax
    int threadId = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;

    if (threadId < maxCells)
    {
        // Use temporary variables for repeated calculations and improve readability
        float x = pointsCoordinates[threadId * 3];
        float y = pointsCoordinates[threadId * 3 + 1];
        float z = pointsCoordinates[threadId * 3 + 2];

        float halfSide = activityFlag[threadId] == 0 ? 0.00f : 0.50f * textureSide;

        int offset36 = quadOffset + threadId * 36;
        int offset24 = quadOffset + maxCells * 4 * 3 * 3 + threadId * 24;
        float textureAbsLength = static_cast<float>(maxCells * textureWidth);
        float textureNormalizedId = static_cast<float>(threadId) * textureWidth / textureAbsLength;
        float textureNormalizedIdNext = static_cast<float>(threadId + 1) * textureWidth / textureAbsLength;

        // Precompute index offsets
        // Vertical x-alligned
        vertexData[offset36] = x - halfSide;
        vertexData[offset36 + 1] = y + halfSide;
        vertexData[offset36 + 2] = z;
        vertexData[offset24] = textureNormalizedId;
        vertexData[offset24 + 1] = 0.00f;

        vertexData[offset36 + 3] = x - halfSide;
        vertexData[offset36 + 4] = y - halfSide;
        vertexData[offset36 + 5] = z;
        vertexData[offset24 + 2] = textureNormalizedId;
        vertexData[offset24 + 3] = 1.00f;

        vertexData[offset36 + 6] = x + halfSide;
        vertexData[offset36 + 7] = y - halfSide;
        vertexData[offset36 + 8] = z;
        vertexData[offset24 + 4] = textureNormalizedIdNext;
        vertexData[offset24 + 5] = 1.00f;

        vertexData[offset36 + 9] = x + halfSide;
        vertexData[offset36 + 10] = y + halfSide;
        vertexData[offset36 + 11] = z;
        vertexData[offset24 + 6] = textureNormalizedIdNext;
        vertexData[offset24 + 7] = 0.00f;

        // Horizontal
        vertexData[offset36 + 12] = x - halfSide;
        vertexData[offset36 + 13] = y;
        vertexData[offset36 + 14] = z + halfSide;
        vertexData[offset24 + 8] = textureNormalizedId;
        vertexData[offset24 + 9] = 1.00f;

        vertexData[offset36 + 15] = x - halfSide;
        vertexData[offset36 + 16] = y;
        vertexData[offset36 + 17] = z - halfSide;
        vertexData[offset24 + 10] = textureNormalizedId;
        vertexData[offset24 + 11] = 0.00f;

        vertexData[offset36 + 18] = x + halfSide;
        vertexData[offset36 + 19] = y;
        vertexData[offset36 + 20] = z - halfSide;
        vertexData[offset24 + 12] = textureNormalizedIdNext;
        vertexData[offset24 + 13] = 0.00f;

        vertexData[offset36 + 21] = x + halfSide;
        vertexData[offset36 + 22] = y;
        vertexData[offset36 + 23] = z + halfSide;
        vertexData[offset24 + 14] = textureNormalizedIdNext;
        vertexData[offset24 + 15] = 1.00f;

        // Vertical z-alligned
        vertexData[offset36 + 24] = x;
        vertexData[offset36 + 25] = y - halfSide;
        vertexData[offset36 + 26] = z + halfSide;
        vertexData[offset24 + 16] = textureNormalizedIdNext;
        vertexData[offset24 + 17] = 1.00f;

        vertexData[offset36 + 27] = x;
        vertexData[offset36 + 28] = y - halfSide;
        vertexData[offset36 + 29] = z - halfSide;
        vertexData[offset24 + 18] = textureNormalizedId;
        vertexData[offset24 + 19] = 1.00f;

        vertexData[offset36 + 30] = x;
        vertexData[offset36 + 31] = y + halfSide;
        vertexData[offset36 + 32] = z - halfSide;
        vertexData[offset24 + 20] = textureNormalizedId;
        vertexData[offset24 + 21] = 0.00f;

        vertexData[offset36 + 33] = x;
        vertexData[offset36 + 34] = y + halfSide;
        vertexData[offset36 + 35] = z + halfSide;
        vertexData[offset24 + 22] = textureNormalizedIdNext;
        vertexData[offset24 + 23] = 0.00f;
    }
}