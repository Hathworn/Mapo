#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Matrix_cos_naive(const float * A, int Acount, int Acols, float * out0, int out0count)
{
    // Calculate unique thread index
    int id = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * gridDim.x * blockDim.x;

    // Perform operation if id within bounds
    if (id < out0count) {
        out0[id] = cosf(A[id]); // Use fast math version of cosine
    }
}