#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_MultiplElementWise_optimized(const float *A, int Acount, int Acols, const float *B, int Bcount, int Bcols, float *out0, int out0count, int out0cols) 
{
    // Simplified thread ID calculation
    int id = blockIdx.x * blockDim.x + threadIdx.x; 

    // Ensure valid index
    if (id < Acount) 
    {
        if (Acount == Bcount) // matrix .* matrix
        {
            out0[id] = A[id] * B[id];
        } 
        else if (Bcols == 1) // matrix .* row vector
        {
            int id_row = id / Acols; // Calculate row index
            out0[id] = A[id] * B[id_row];
        } 
        else // matrix .* column vector
        {
            int id_col = id % Acols; // Calculate column index
            out0[id] = A[id] * B[id_col];
        }
    }
}