#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeErrorPerWinningKernel(float *localError, int *winningCount, float *errorPerWinning, int *activityFlag, int maxCells)
{
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    // Simplified logic using sequential conditions
    if (threadId < maxCells && activityFlag[threadId] == 1)
    {
        errorPerWinning[threadId] = (winningCount[threadId] != 0) ? localError[threadId] / (float)winningCount[threadId] : 0.0f;
    }
}