#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ChangeOutputWeightsKernel( float *outputWeights, float *outputWeightDeltas, float *outputDeltas, float *hiddenActivations,  float trainingRate, float momentum )
{
    // Compute a unique weightId using a 2D grid and 3D block
    int weightId = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Using integer division and modulus to calculate indices to and from
    int to = weightId / D_HIDDEN_UNITS; 
    int from = weightId % D_HIDDEN_UNITS;

    // Check if within bounds
    if (weightId < D_OUTPUT_UNITS * D_HIDDEN_UNITS) 
    {
        // Compute the gradient and weight delta, then update deltas and weights
        float gradient = outputDeltas[to] * hiddenActivations[from];
        float weightDelta = trainingRate * gradient + momentum * outputWeightDeltas[weightId];
        outputWeightDeltas[weightId] = weightDelta;
        outputWeights[weightId] += weightDelta;
    }
}