#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void RoundKernel_optimized(const float *A, int Acount, int Acols, float *out0, int out0count)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread ID calculation
    if (id < out0count)
    {
        out0[id] = roundf(A[id]); // Use faster roundf function for float
    }
}