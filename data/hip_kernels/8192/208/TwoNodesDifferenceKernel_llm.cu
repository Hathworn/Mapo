#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TwoNodesDifferenceKernel(int nodeOne, int nodeTwo, int vectorLength, float *referenceVector, float *twoNodesDifference)
{
    // Calculate unique thread ID using 1D grid
    int threadId = blockDim.x * blockIdx.x + threadIdx.x;

    // Process only if threadId is within bounds
    if(threadId < vectorLength)
    {
        // Compute index offsets directly
        int indexOne = nodeOne * vectorLength + threadId;
        int indexTwo = nodeTwo * vectorLength + threadId;
        
        // Calculate difference
        twoNodesDifference[threadId] = referenceVector[indexOne] - referenceVector[indexTwo];
    }
}