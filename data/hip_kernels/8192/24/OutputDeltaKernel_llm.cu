#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void OutputDeltaKernel(float *outputDeltas, float *target, float *outputActivations, float *outputActivationDerivatives)
{
    int unitId = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified unitId calculation
    if (unitId < D_OUTPUT_UNITS)
    {
        outputDeltas[unitId] = (target[unitId] - outputActivations[unitId]) * outputActivationDerivatives[unitId];
    }
}