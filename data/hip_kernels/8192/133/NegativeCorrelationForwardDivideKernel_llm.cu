#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NegativeCorrelationForwardDivideKernel(float* outputPtr, int thisLayerSize, int inputModelCount)
{
    // Calculate global thread index
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (j < thisLayerSize)
    {
        // Divide the output by inputModelCount for the corresponding element
        outputPtr[j] /= static_cast<float>(inputModelCount);
    }
}