#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InterpolateSymbolsKernel(float *symbolVectors, int symbolOneId, int symbolTwoId, float weightOne, float weightTwo, float *resultSymbol, int symbolSize)
{
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate only necessary thread operations
    if (threadId < symbolSize)
    {
        int symbolOneCellId = symbolOneId * symbolSize + threadId;
        int symbolTwoCellId = symbolTwoId * symbolSize + threadId;

        resultSymbol[threadId] = weightOne * symbolVectors[symbolOneCellId] + weightTwo * symbolVectors[symbolTwoCellId];
    }
}