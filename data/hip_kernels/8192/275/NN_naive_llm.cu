#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float Dist_between_two_vec(float * v0, float *v1, int size) {
    float dist = 0;
    for (int i = 0; i < size; i++)
        dist += (v0[i] - v1[i]) * (v0[i] - v1[i]);
    return sqrt(dist);
}

__global__ void NN_naive(float * A, int colsA, int sizeA, float * B, int colsB, int numsB, int dim, float * idx, float * dist) {
    float tmp_dist = FLT_MAX; // Use FLT_MAX for clarity and accuracy
    int nn_id = -1;
    int idA = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    if (idA >= sizeA) return; // Early exit for invalid threads

    for (int idB = 0; idB < numsB; idB++) { // Use actual count of B vectors
        float adist = Dist_between_two_vec(A + colsA * idA, B + colsB * idB, dim);
        if (tmp_dist > adist) {
            tmp_dist = adist;
            nn_id = idB;
        }
    }
    if (idA < sizeA) { // Check bounds before storing results
        dist[idA] = tmp_dist;
        idx[idA] = nn_id;
    }
}