#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SumBasicSymbolsKernel(float *symbolVectors, int symbolOneId, int symbolTwoId, float *result, int symbolSize)
{
    // Use more concise expression for computing 'threadId'
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x; 

    // Use shared memory to reduce global memory access
    __shared__ float sharedData[blockDim.x];

    if(threadId < symbolSize)
    {
        int offsetOne = symbolOneId * symbolSize + threadId;
        int offsetTwo = symbolTwoId * symbolSize + threadId;
        sharedData[threadIdx.x] = symbolVectors[offsetOne] + symbolVectors[offsetTwo];
        __syncthreads(); // Ensure all threads have completed their operations

        // Write result back to global memory
        result[threadId] = sharedData[threadIdx.x];
    }
}