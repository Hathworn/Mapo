#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__  void DecodeValues(float* superposition, int symbolSize, float* output, float* reliability, int numOfValues, int squaredMode, float* dirX, float* dirY, float* negDirX, float* negDirY, float* originX, float* originY)
{
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    if (threadId >= numOfValues)
        return;

    float* dir = (threadId % 2 == 0) ? dirX : dirY;
    float* negDir = (threadId % 2 == 0) ? negDirX : negDirY;
    float* origin = (threadId % 2 == 0) ? originX : originY;

    float outVal = 0;
    float relVal = 0;

    // Using float local variables for accumulation to optimize memory access
    for (int i = 0; i < symbolSize; i++)
    {
        float sup_i = superposition[i];
        outVal += sup_i * (dir[i] - negDir[i]);  // Combined computation to reduce redundant operations
        relVal += sup_i * origin[i];
    }

    // Final calculations
    relVal += fabs(outVal);
    output[threadId] = outVal / relVal;
    reliability[threadId] = relVal;
}