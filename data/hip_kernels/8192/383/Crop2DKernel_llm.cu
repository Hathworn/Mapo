#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Crop2DKernel(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int size, int leftMargin, int topMargin, float fillValue)
{
    // Calculate flattened global index
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    if (id >= size) return; // Exit early if out of bounds

    // Calculate 2D coordinates in the output image
    int outputX = id % outputWidth;
    int outputY = id / outputWidth;

    // Map to 2D coordinates in the input image
    int inputX = outputX - leftMargin;
    int inputY = outputY - topMargin;

    // Check if input coordinates are valid
    if (inputX >= 0 && inputX < inputWidth && inputY >= 0 && inputY < inputHeight)
        output[id] = input[inputX + inputY * inputWidth];
    else
        output[id] = fillValue;
}