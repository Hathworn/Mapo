#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SumNewCentroidCoordinatesKernel(float *input, int imgWidth, int imgHeight, float *centroidCoordinates, int *nearestCentroid, float *pointsWeight, int inputSize)
{
    // Calculate unique thread ID
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread processes a valid point
    if (threadId < inputSize)
    {
        // Calculate x and y coordinates from thread ID
        int pointX = threadId % imgWidth;
        int pointY = threadId / imgWidth;

        float X = static_cast<float>(pointX);
        float Y = static_cast<float>(pointY);

        int centroidId = nearestCentroid[threadId];

        float weight = input[threadId];

        // Use atomic operations to update shared data
        atomicAdd(&centroidCoordinates[centroidId * 2], weight * X);
        atomicAdd(&centroidCoordinates[centroidId * 2 + 1], weight * Y);
        atomicAdd(&pointsWeight[centroidId], weight);
    }
}