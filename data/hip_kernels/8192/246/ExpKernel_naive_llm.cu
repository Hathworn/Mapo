#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ExpKernel_optimized(const float *A, int Acount, int Acols, float *out0, int out0count)
{
    // Calculate a flat index based on 3D grid and block dimensions
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    if (id < out0count)
    {
        // Efficient usage of memory by directly accessing index and computing the exponential
        out0[id] = expf(A[id]);
    }
}