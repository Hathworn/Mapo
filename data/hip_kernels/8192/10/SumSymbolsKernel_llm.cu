#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SumSymbolsKernel( float *symbolOne, float *symbolTwo, float *result, int symbolSize )
{
    // Optimize thread index calculation using grid-stride loop to improve utilization
    for (int threadId = blockIdx.x * blockDim.x + threadIdx.x; 
         threadId < symbolSize; 
         threadId += blockDim.x * gridDim.x)
    {
        result[threadId] = symbolOne[threadId] + symbolTwo[threadId];
    }
}