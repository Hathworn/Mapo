#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InputWeightsRTRLDerivativesKernel(float *input, float *hiddenActivationDerivatives, float *recurrentWeights, float *inputWeightRTRLDerivatives, float *previousInputWeightRTRLDerivatives)
{
    int partialId = blockDim.x * blockIdx.y * gridDim.x // rows preceding current row in grid
                  + blockDim.x * blockIdx.x            // blocks preceding current block
                  + threadIdx.x;

    if (partialId < D_HIDDEN_UNITS * D_HIDDEN_UNITS * D_INPUT_UNITS)
    {
        int unitId = partialId / (D_HIDDEN_UNITS * D_INPUT_UNITS);
        int weightId = partialId % (D_HIDDEN_UNITS * D_INPUT_UNITS);
        int to = weightId / D_INPUT_UNITS;
        int from = weightId % D_INPUT_UNITS;

        // Use shared memory to reduce global memory access
        __shared__ float sharedRecurrentWeights[D_HIDDEN_UNITS];
        __shared__ float sharedPreviousDerivatives[D_HIDDEN_UNITS];
        
        float sum = 0;
        for (int i = 0; i < D_HIDDEN_UNITS; i++)
        {
            // Load values into shared memory
            if (threadIdx.x < D_HIDDEN_UNITS)
            {
                sharedRecurrentWeights[threadIdx.x] = recurrentWeights[unitId * D_HIDDEN_UNITS + threadIdx.x];
                sharedPreviousDerivatives[threadIdx.x] = previousInputWeightRTRLDerivatives[threadIdx.x * (D_HIDDEN_UNITS * D_INPUT_UNITS) + weightId];
            }
            __syncthreads(); // Synchronize to ensure all threads have loaded their data

            sum += sharedRecurrentWeights[i] * sharedPreviousDerivatives[i];
        }

        inputWeightRTRLDerivatives[partialId] = hiddenActivationDerivatives[unitId] * ((unitId == to) * input[from] + sum);
    }
}