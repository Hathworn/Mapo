#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyKernel(float* from, float* to, int size)
{
    // Optimize thread id calculation using 1D grid
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < size)
    {
        to[threadId] = from[threadId];
    }
}