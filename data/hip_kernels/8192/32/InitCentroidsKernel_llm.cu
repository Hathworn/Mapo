#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InitCentroidsKernel(float *centroidCoordinates, float *randomNumbers, float minX, float maxX, float minY, float maxY, int centroids)
{
    // Calculate the global thread ID more efficiently
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Process only if the threadId is within valid range
    if (threadId < centroids)
    {
        // Compute the x and y coordinates for the centroids
        int index = threadId * 2; // Reduce redundant calculations
        centroidCoordinates[index] = (maxX - minX) * randomNumbers[index] + minX;
        centroidCoordinates[index + 1] = (maxY - minY) * randomNumbers[index + 1] + minY;
    }
}