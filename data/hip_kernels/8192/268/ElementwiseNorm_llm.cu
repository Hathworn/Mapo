#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ElementwiseNorm(float * A, float * B, int size) {
    // Calculate the global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process multiple elements per thread to improve memory access
    for (int i = id; i < size; i += blockDim.x * gridDim.x) {
        A[i] /= B[i];
    }
}