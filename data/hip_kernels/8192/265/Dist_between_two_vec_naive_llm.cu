#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float Dist_between_two_vec(float *v0, float *v1, int size) {
    float dist = 0;
    for (int i = 0; i < size; i++)
        dist += (v0[i] - v1[i]) * (v0[i] - v1[i]);
    return sqrt(dist);
}

__global__ void Dist_between_two_vec_optimized(float *v0, float *v1, int size, float *dst) {
    __shared__ float shared_dist[256];  // Use shared memory for partial results
    int idx = threadIdx.x;
    shared_dist[idx] = 0;

    for (int i = idx; i < size; i += blockDim.x) {
        shared_dist[idx] += (v0[i] - v1[i]) * (v0[i] - v1[i]); 
    }

    __syncthreads();

    // Reduction to calculate the sum of squares
    if (idx == 0) {
        float dist = 0;
        for (int i = 0; i < blockDim.x; i++) {
            dist += shared_dist[i];
        }
        dst[0] = dist; 
    }
}