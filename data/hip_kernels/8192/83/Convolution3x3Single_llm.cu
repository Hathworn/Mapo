#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float mulWithKernel(int x, int y, int kx, int ky, float* input, int width, int height)
{
    int px = min(max(x, 0), width - 1);
    int py = min(max(y, 0), height - 1);
    
    return D_KERNEL[3 * (ky + 1) + kx + 1] * input[py * width + px];
}

__global__ void Convolution3x3Single(float* input, float* output, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within the bounds
    if (x < width && y < height)
    {
        float result = 0;

        // Unrolling the loops to reduce overhead
        #pragma unroll
        for (int dy = -1; dy <= 1; ++dy)
        {
            #pragma unroll
            for (int dx = -1; dx <= 1; ++dx)
            {
                result += mulWithKernel(x + dx, y + dy, dx, dy, input, width, height);
            }
        }

        output[y * width + x] = result;
    }
}