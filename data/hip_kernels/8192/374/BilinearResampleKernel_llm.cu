#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BilinearResampleKernel(float *input, float *output, int inputWidth, int inputHeight, int outputWidth, int outputHeight)
{
    int size = outputWidth * outputHeight;
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x; // Optimize thread id calculation

    if (id < size) {
        int px = id % outputWidth;
        int py = id / outputWidth;

        float xRatio = (float)(inputWidth - 1) / (outputWidth - 1);
        float yRatio = (float)(inputHeight - 1) / (outputHeight - 1);

        float x = xRatio * px;
        float y = yRatio * py;

        int xL = (int)x; // Use (int) directly for floor
        int xR = min(xL + 1, inputWidth - 1); // Use min to prevent array bounds exceed
        int yT = (int)y; // Use (int) directly for floor
        int yB = min(yT + 1, inputHeight - 1); // Use min to prevent array bounds exceed
        
        float dL = xR - x; // Simplify distance calculations
        float dR = 1.0f - dL;
        float dT = yB - y;
        float dB = 1.0f - dT;

        float topLeft = input[yT * inputWidth + xL];
        float topRight = input[yT * inputWidth + xR];
        float bottomLeft = input[yB * inputWidth + xL];
        float bottomRight = input[yB * inputWidth + xR];

        float iT = (xL == xR) ? topLeft : topLeft * dL + topRight * dR; // Use ternary operator for conditional assignment
        float iB = (xL == xR) ? bottomLeft : bottomLeft * dL + bottomRight * dR;

        output[py * outputWidth + px] = (yT == yB) ? iT : iT * dT + iB * dB; // Use ternary operator for conditional assignment
    }
}