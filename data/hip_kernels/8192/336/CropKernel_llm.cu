#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CropKernel(float min, float max, float* input, float* output, int size)
{
    // Optimize id calculation for better memory access pattern
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id < size)
    {
        // Inline function call for efficiency
        output[id] = fmaxf(fminf(input[id], max), min);
    }
}