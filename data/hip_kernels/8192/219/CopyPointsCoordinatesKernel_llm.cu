#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CopyPointsCoordinatesKernel(float *pointsCoordinates, int *activityFlag, float xNonValid, float yNonValid, float zNonValid, float *dataVertex, int dataVertexOffset, int maxCells)
{
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x; // Simplified thread ID calculation

    if(threadId < maxCells)
    {
        int baseIdx = threadId * 3; // Precompute index to avoid repetition
        float xToCopy = pointsCoordinates[baseIdx];
        float yToCopy = pointsCoordinates[baseIdx + 1];
        float zToCopy = pointsCoordinates[baseIdx + 2];
        
        if(activityFlag[threadId] == 0) // Check activity flag
        {
            xToCopy = xNonValid;
            yToCopy = yNonValid;
            zToCopy = zNonValid;
        }
        
        int dataIdx = dataVertexOffset + baseIdx; // Use precomputed index for dataVertex
        dataVertex[dataIdx] = xToCopy;
        dataVertex[dataIdx + 1] = yToCopy;
        dataVertex[dataIdx + 2] = zToCopy;
    }
}