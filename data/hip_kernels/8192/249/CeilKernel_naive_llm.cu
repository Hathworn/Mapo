#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CeilKernel_naive(const float *A, int Acount, int Acols, float *out0, int out0count)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified id calculation
    int stride = blockDim.x * gridDim.x;            // Calculate stride for grid-stride loop
    for (int i = id; i < out0count; i += stride)    // Use grid-stride loop to process elements
    {
        out0[i] = ceil(A[i]);
    }
}