#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SetValue(float* im, float val, int size)
{
    // Use a single calculation for the thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x + 
             blockIdx.y * blockDim.x * gridDim.x;
    // Ensure in-bounds access
    if (id < size)
    {
        im[id] = val;
    }
}