#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize indexing calculation
__global__ void OutputDeltaKernel(float *outputDeltas, float *target, float *outputActivations, float *outputActivationDerivatives)
{
    int unitId = blockIdx.x * blockDim.x + threadIdx.x; // Simplified indexing with flat structure
    if (unitId < D_OUTPUT_UNITS)
    {
        outputDeltas[unitId] = (target[unitId] - outputActivations[unitId]) * outputActivationDerivatives[unitId];
    }
}