#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void oneOfNSelection(float *buffer, int* index, int size, float value)
{
    // Calculate a unique global thread ID
    int threadId = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    if (threadId < size) {
        // Set buffer[threadId] to value if threadId matches index[0], otherwise set to 0
        buffer[threadId] = (threadId == index[0]) ? value : 0;
    }
}