#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatMultipl_optimized(float * A, float * B, float * C, int nColsA, int nColsB, int sizeC) {
    int i_col = blockIdx.x * blockDim.x + threadIdx.x; // index in row
    int i_row = blockIdx.y * blockDim.y + threadIdx.y; // index in column
    int idx = i_row * nColsB + i_col; // # of cols in B = # of cols in C

    if (i_row < sizeC / nColsB && i_col < nColsB) { // Boundary check optimization
        float Cvalue = 0;
        for (int e = 0; e < nColsA; e++) {
            Cvalue += A[i_row * nColsA + e] * B[e * nColsB + i_col];
        }
        C[idx] = Cvalue;
    }
}