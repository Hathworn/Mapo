#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void InvolveVector(float* input, float* output, int inputSize)
{
    int threadId = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    // Use a single thread to copy the first element outside of the loop
    if (threadId == 0) 
    {
        output[0] = input[0];
    }

    // Reverse input into output using multiple threads efficiently
    if (threadId < inputSize - 1) 
    {
        output[threadId + 1] = input[inputSize - threadId - 1];
    }
}