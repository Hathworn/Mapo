#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FloorKernel_optimized(const float * A, int Acount, int Acols, float * out0, int out0count)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Use grid-stride loop for better parallelism
    for (int i = id; i < out0count; i += blockDim.x * gridDim.x) 
    {
        out0[i] = floor(A[i]);
    }
}