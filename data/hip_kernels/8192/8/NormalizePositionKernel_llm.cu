#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizePositionKernel(float *input, float *normalized, float xMax, float yMax) 
{
    // Calculate a unique global thread ID for the 1D input
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Only the first thread performs normalization
    if (threadId == 0)
    {
        normalized[0] = input[0] / xMax;
        normalized[1] = input[1] / yMax;
    }
}