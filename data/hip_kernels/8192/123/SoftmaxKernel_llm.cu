#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SoftmaxKernel(float *outputPtr, float expSum, int layerSize) {
    // Calculate global thread index using 2D grid for better dimensionality 
    int i = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;

    // If the global index is within bounds, normalize the softmax values
    if (i < layerSize) {
        outputPtr[i] /= expSum;
    }
}