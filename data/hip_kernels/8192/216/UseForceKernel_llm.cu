#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UseForceKernel(float *force, float forceFactor, float *pointsCoordinates, int maxCells)
{
    // Calculate global thread ID efficiently
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadId < maxCells * 3)
    {
        // Perform operation by accessing memory efficiently
        pointsCoordinates[threadId] += forceFactor * force[threadId];
    }
}