#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_getCol_FloatId_naive(const float * A, int Acount, int Acols, float * out0, int out0count, int out0cols, float col_id)
{
    // Compute unique global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop over rows with thread stride pattern for better utilization
    for (int row = id; row < Acount / Acols; row += blockDim.x * gridDim.x)
    {
        out0[row] = A[row * Acols + (int)col_id];
    }
}