#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LengthFromElements(float* element1, float* element2, float* output, int count)
{
    // Calculate unique global thread ID
    int threadId = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Ensure the thread ID is within the bounds
    if(threadId < count)
    {
        // Compute the length and store in the output
        float x = element1[threadId];
        float y = element2[threadId];
        output[threadId] = sqrtf(x * x + y * y);
    }
}