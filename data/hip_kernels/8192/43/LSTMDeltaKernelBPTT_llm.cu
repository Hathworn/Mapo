#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void LSTMDeltaKernelBPTT( float* deltas, float* cellStates, float* previousCellStates, float* cellStateErrors, float* nextCellStateErrors,  float* outputGateDeltas, float* forgetGateDeltas, float* nextForgetGateDeltas, float* inputGateDeltas, float* nextInputGateDeltas, float* cellInputDeltas,  float* cellInputActivations, float* cellStateActivations, float* outputGateActivations, float* nextForgetGateActivations, float* inputGateActivations,  float* cellInputActivationDerivatives, float* cellStateActivationDerivatives, float* outputGateActivationDerivatives, float* forgetGateActivationDerivatives, float* inputGateActivationDerivatives,  float* cellInputWeights, float* outputGateWeights, float* forgetGateWeights, float* inputGateWeights,  int inputCount, int cellCount, int cellsPerBlock )
{
    int memoryBlockId = blockDim.x * blockIdx.y * gridDim.x  // rows preceding current row in grid
                      + blockDim.x * blockIdx.x             // blocks preceding current block
                      + threadIdx.x;

    if (memoryBlockId < cellCount / cellsPerBlock)
    {
        // Optimize the computation by reducing redundant calculations
        int blockStart = memoryBlockId * cellsPerBlock;
        int blockEnd = (memoryBlockId + 1) * cellsPerBlock;
        int peepHoleBaseId = memoryBlockId * (inputCount + cellCount + cellsPerBlock + 1) + inputCount + cellCount;

        float outputDeltaSum = 0.0f;
        for (int cellId = blockStart; cellId < blockEnd; cellId++)
        {
            outputDeltaSum += cellStateActivations[cellId] * deltas[cellId];
        }
        outputGateDeltas[memoryBlockId] = outputDeltaSum * outputGateActivationDerivatives[memoryBlockId];

        // Reuse common calculations to minimize operations inside the loop
        float outputActivation = outputGateActivations[memoryBlockId];
        float nextForgetActivation = nextForgetGateActivations[memoryBlockId];
        float nextInputDelta = nextInputGateDeltas[memoryBlockId];
        float nextForgetDelta = nextForgetGateDeltas[memoryBlockId];
        float outputDelta = outputGateDeltas[memoryBlockId];
        float inputActivation = inputGateActivations[memoryBlockId];

        for (int cellId = blockStart; cellId < blockEnd; cellId++)
        {
            int relativeCellId = cellId - blockStart;
            int peepHoleWeightId = peepHoleBaseId + relativeCellId;

            float cellStateErr = deltas[cellId] * outputActivation * cellStateActivationDerivatives[cellId] +
                                 nextCellStateErrors[cellId] * nextForgetActivation +
                                 nextInputDelta * inputGateWeights[peepHoleWeightId] +
                                 nextForgetDelta * forgetGateWeights[peepHoleWeightId] +
                                 outputDelta * outputGateWeights[peepHoleWeightId];
            cellStateErrors[cellId] = cellStateErr;

            cellInputDeltas[cellId] = inputActivation * cellInputActivationDerivatives[cellId] * cellStateErr;
        }

        float inputDeltaSum = 0.0f;
        float forgetDeltaSum = 0.0f;
        for (int cellId = blockStart; cellId < blockEnd; cellId++)
        {
            inputDeltaSum += cellStateErrors[cellId] * cellInputActivations[cellId];
            forgetDeltaSum += cellStateErrors[cellId] * previousCellStates[cellId];
        }
        inputGateDeltas[memoryBlockId] = inputDeltaSum * inputGateActivationDerivatives[memoryBlockId];
        forgetGateDeltas[memoryBlockId] = forgetDeltaSum * forgetGateActivationDerivatives[memoryBlockId];
    }
}