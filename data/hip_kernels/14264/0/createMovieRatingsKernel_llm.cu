#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void createMovieRatingsKernel(const float *weights, const float *initial_hidden_feature_probs, float* movie_rating_probs, int num_movies, int num_hidden_features) {

    unsigned int movie_rating_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop over all movie_id/rating_id pairs handled by this thread
    while (movie_rating_id < num_movies * 5) {
        float dot_prod = 0.0f; // Initialize dot product

        // Unrolled loop for optimizing dot product calculation
        for (unsigned int hidden_id = 0; hidden_id < num_hidden_features; hidden_id += 4) {
            dot_prod += weights[movie_rating_id * num_hidden_features + hidden_id] * initial_hidden_feature_probs[hidden_id];
            if (hidden_id + 1 < num_hidden_features) {
                dot_prod += weights[movie_rating_id * num_hidden_features + hidden_id + 1] * initial_hidden_feature_probs[hidden_id + 1];
            }
            if (hidden_id + 2 < num_hidden_features) {
                dot_prod += weights[movie_rating_id * num_hidden_features + hidden_id + 2] * initial_hidden_feature_probs[hidden_id + 2];
            }
            if (hidden_id + 3 < num_hidden_features) {
                dot_prod += weights[movie_rating_id * num_hidden_features + hidden_id + 3] * initial_hidden_feature_probs[hidden_id + 3];
            }
        }

        // Store result
        movie_rating_probs[movie_rating_id] = dot_prod;

        // Move to the next point
        movie_rating_id += blockDim.x * gridDim.x;
    }
}