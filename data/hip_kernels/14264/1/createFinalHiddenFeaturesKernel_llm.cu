#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void createFinalHiddenFeaturesKernel(const float *weights, const float *movie_rating_probs, float* final_hidden_feature_probs, int num_movies, int num_hidden_features) {

    // Calculate unique global thread index
    unsigned int hidden_id = blockIdx.x * blockDim.x + threadIdx.x;
    float dot_prod; // Temporary, local dot product variable

    // Ensure the thread works until all the hidden features are processed
    while (hidden_id < num_hidden_features) {
        dot_prod = 0.0f; // Reset the dot product to 0

        // Iterate over each movie
        for (int movie_id = 0; movie_id < num_movies; movie_id++) {
            // Vectorized dot product computation for movie ratings
            #pragma unroll
            for (int rating = 0; rating < 5; rating++) {
                dot_prod += weights[movie_id * 5 * num_hidden_features + rating * num_hidden_features + hidden_id]
                            * movie_rating_probs[movie_id * 5 + rating];
            }
        }

        // Store the dot_product result
        final_hidden_feature_probs[hidden_id] = dot_prod;

        // Move to the next element of the hidden features for the current thread
        hidden_id += blockDim.x * gridDim.x;
    }
}