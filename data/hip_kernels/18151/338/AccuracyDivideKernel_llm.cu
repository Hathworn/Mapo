#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load value from global memory to register
    float acc = *accuracy;

    // Perform division in register
    acc /= N;

    // Write back result from register to global memory
    *accuracy = acc;
}