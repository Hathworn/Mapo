#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"


__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn){
    double out = 0.0;  // Initialize output to zero before loop
    long i = threadIdx.x + blockIdx.x * blockDim.x;  // Calculate global thread ID
    if (i < dim_mn) {
        #pragma unroll  // Encourage loop unrolling for better performance
        for (long j = 0; j < dim_mn; j++) {
            out += matrix[i * dim_mn + j] * vector_in[j];
        }
        vector_out[i] = out;  // Store result in output vector
    }
}