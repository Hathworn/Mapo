#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn){
    extern __shared__ double s_vector_in[]; // Use shared memory for vector
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Load vector elements into shared memory
    long j = threadIdx.x;
    if (j < dim_mn) {
        s_vector_in[j] = vector_in[j];
    }
    __syncthreads(); // Ensure all data is loaded

    if (i < dim_mn) {
        double out = 0.0;
        // Use shared memory for input vector and do computation
        for (j = 0; j < dim_mn; j++) {
            out += matrix[i * dim_mn + j] * s_vector_in[j];
        }
        vector_out[i] = out;
    }
}