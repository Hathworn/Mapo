#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Use blockIdx and blockDim to allow for larger vectors
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Avoid out-of-bounds access
    if (i < N) {  // Assuming N is defined as the length of the vectors
        c[i] = a[i] + b[i];
    }
}