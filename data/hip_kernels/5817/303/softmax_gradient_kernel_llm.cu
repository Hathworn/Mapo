#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    extern __shared__ float reduction_buffer[]; // Utilize dynamic shared memory
    const int idx = threadIdx.x;
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;

    // Efficient parallel reduction
    float tmp = 0;
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();
    
    // Reduce using binary tree reduction pattern
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }
    
    // Compute gradient using the reduced inner product
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}