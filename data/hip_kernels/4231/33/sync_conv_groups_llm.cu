#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups() {
    // Use cooperative groups to manage synchronization more efficiently
    // Obtain the grid group for synchronization
    auto gridGroup = cooperative_groups::this_grid();
    
    // Synchronize all threads in the grid
    gridGroup.sync();
}