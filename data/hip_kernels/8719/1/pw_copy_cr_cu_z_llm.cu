#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_copy_cr_cu_z(const double *zin, double *dout, const int n) {
    // Calculate global index with grid-stride loop for better performance
    for (int igpt = blockIdx.x * blockDim.x + threadIdx.x;
         igpt < n;
         igpt += blockDim.x * gridDim.x) {
        dout[igpt] = zin[2 * igpt];
    }
}