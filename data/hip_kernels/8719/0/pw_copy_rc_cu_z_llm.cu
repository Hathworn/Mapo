#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_copy_rc_cu_z(const double *din, double *zout, const int n) {
    // Use a single calculation for the global thread index
    int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Ensure the thread operates only on valid data
    if (igpt < n) {
        zout[2 * igpt] = din[igpt];
        zout[2 * igpt + 1] = 0.0;
    }
}