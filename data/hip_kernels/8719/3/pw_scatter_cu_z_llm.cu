#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_scatter_cu_z(double *c, const double *pwcc, const double scale, const int ngpts, const int nmaps, const int *ghatmap) {
    // Calculate global thread index more efficiently using built-in variable
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Use a simpler check structure to enhance readability
    if (igpt >= ngpts) return;

    // Reduced array lookups and operations in the main routine
    int mapIndex = ghatmap[igpt];
    int srcIndex = 2 * igpt;
    int destIndex = 2 * mapIndex;
    c[destIndex] = scale * pwcc[srcIndex];
    c[destIndex + 1] = scale * pwcc[srcIndex + 1];

    // Handle the nmaps == 2 case, optimizing similar patterns
    if (nmaps == 2) {
        mapIndex = ghatmap[igpt + ngpts];
        destIndex = 2 * mapIndex;
        c[destIndex] = scale * pwcc[srcIndex];
        c[destIndex + 1] = -scale * pwcc[srcIndex + 1];
    }
}