#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_gather_cu_z(      double *pwcc, const double *c, const double  scale, const int     ngpts, const int    *ghatmap) {
    // Calculate unique thread index across all blocks
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    if (igpt < ngpts) {
        int mappedIdx = ghatmap[igpt];
        // Directly access the mapped index to minimize memory indexing overhead
        pwcc[2 * igpt    ] = scale * c[2 * mappedIdx    ];
        pwcc[2 * igpt + 1] = scale * c[2 * mappedIdx + 1];
    }
}