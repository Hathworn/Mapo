#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

__global__ void variance(int* n, double* x, double* mean)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensuring index is within bounds
    if (i < *n) {
        double diff = x[i] - *mean;
        x[i] = diff * diff;
    }
}