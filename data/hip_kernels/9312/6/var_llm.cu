#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void var(int *a, int *b, int n, float mean)
{
    int tid = threadIdx.x;
    int block = 256 * blockIdx.x;
    __shared__ float shared_sum[256]; // Use shared memory

    float local_sum = 0.0f;

    // Unroll loop for performance
    for (int i = block + tid; i < min(block + 256, n); i += 256)
    {
        float diff = a[i] - mean;
        local_sum += diff * diff;
    }

    // Store each thread's local sum in shared memory
    shared_sum[tid] = local_sum;
    __syncthreads(); // Ensure all threads have written their local sum

    // Reduce within the block using parallel reduction
    if (tid < 128) shared_sum[tid] += shared_sum[tid + 128];
    __syncthreads();
    if (tid < 64) shared_sum[tid] += shared_sum[tid + 64];
    __syncthreads();
    if (tid < 32)
    {
        shared_sum[tid] += shared_sum[tid + 32];
        shared_sum[tid] += shared_sum[tid + 16];
        shared_sum[tid] += shared_sum[tid + 8];
        shared_sum[tid] += shared_sum[tid + 4];
        shared_sum[tid] += shared_sum[tid + 2];
        shared_sum[tid] += shared_sum[tid + 1];
    }

    // First thread writes the block's result
    if (tid == 0)
    {
        b[blockIdx.x] = shared_sum[0];
    }
}