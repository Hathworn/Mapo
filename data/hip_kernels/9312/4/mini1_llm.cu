#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mini1(int *a, int *b, int n)
{
    // Use shared memory for reduced memory access latency
    __shared__ int shared_min[256];

    int local_idx = threadIdx.x;
    int global_idx = blockIdx.x * blockDim.x + local_idx;
    int block_size = blockDim.x;
    int mini = 7888888;

    // Compute the local minimum in parallel
    if (global_idx < n) {
        mini = a[global_idx];
    }
    shared_min[local_idx] = mini;

    // Synchronize to ensure all threads copied their value
    __syncthreads();

    // Reduce to find the local block minimum
    for (int stride = block_size / 2; stride > 0; stride >>= 1) {
        if (local_idx < stride && (global_idx + stride) < n) {
            shared_min[local_idx] = min(shared_min[local_idx], shared_min[local_idx + stride]);
        }
        __syncthreads();
    }

    // Write the block minimum to the output array
    if (local_idx == 0) {
        b[blockIdx.x] = shared_min[0];
    }
}