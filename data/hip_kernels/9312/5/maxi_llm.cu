#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void maxi(int * a, int * b, int n) {

    __shared__ int local_max[256]; // Shared memory for partial maximum

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int local_max_val = (idx < n) ? a[idx] : INT_MIN; // Initialize local max

    // Reduce within block
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        __syncthreads(); // Ensure all threads have written their values
        if (threadIdx.x < stride) {
            int other = local_max_val;
            if (idx + stride < n) other = a[idx + stride];
            local_max_val = max(local_max_val, other);
        }
    }

    // Store the result from the first thread in each block
    if (threadIdx.x == 0) {
        b[blockIdx.x] = local_max_val;
    }
}