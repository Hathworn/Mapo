#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function for summing elements
__global__ void sum(int *a, int *b, int n)
{
    // Shared memory for partial sums
    __shared__ int sharedSum[256];
    
    // Calculate global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize shared memory to zero
    sharedSum[threadIdx.x] = (idx < n) ? a[idx] : 0;
    __syncthreads();

    // Perform sum reduction within a block
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if (threadIdx.x % (2 * stride) == 0) {
            sharedSum[threadIdx.x] += sharedSum[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write the result from the first thread of each block
    if (threadIdx.x == 0) {
        b[blockIdx.x] = sharedSum[0];
    }
}