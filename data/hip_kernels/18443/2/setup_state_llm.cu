#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Use blockIdx.x to allow multiple threads to initialize different states
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}