#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    int32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Each thread calculates its global index and checks boundaries
    if (tid < N) {
        output[tid] = -row_sum[blockIdx.x];
    }
}