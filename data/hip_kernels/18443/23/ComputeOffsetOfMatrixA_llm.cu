#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;  // Compute global thread index
    if (idx < N) {
        output[blockIdx.x * N + idx] = -col_sum[idx];  // Efficient index calculations
    }
}