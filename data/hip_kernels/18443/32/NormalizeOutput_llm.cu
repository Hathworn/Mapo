#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for loop

    for (; idx < num_elements; idx += stride) {
        to_normalize[idx * 3] = batch_index; // Batch index assignment
        to_normalize[idx * 3 + 1] = class_index; // Class index assignment
        to_normalize[idx * 3 + 2] = static_cast<int64_t>(original[idx]); // Original element assignment
    }
}