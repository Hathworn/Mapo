#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate thread's absolute index in the data
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure the thread index is within the bounds
    if (idx < numEls) {
        dest[idx] = a[idx] + b[idx];
    }
}