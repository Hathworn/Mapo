#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_unpack_yuy2_rgba8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (i < superPixCount) {  // Check bounds early to avoid unnecessary checks in the loop
        for (; i < superPixCount; i += stride) {

            int idx = i * 4;

            uint8_t y0 = src[idx];
            uint8_t u0 = src[idx + 1];
            uint8_t y1 = src[idx + 2];
            uint8_t v0 = src[idx + 3];

            int16_t c0 = y0 - 16;
            int16_t c1 = y1 - 16;
            int16_t d = u0 - 128;
            int16_t e = v0 - 128;

            int32_t t;
            #define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

            int odx = i * 8;

            // Calculate common values once to reduce repeated operations
            int32_t redFactor = 298 * c0 + 409 * e + 128;
            int32_t greenFactor = 298 * c0 - 100 * d - 409 * e + 128;
            int32_t blueFactor = 298 * c0 + 516 * d + 128;

            dst[odx] = clamp(redFactor >> 8);
            dst[odx + 1] = clamp(greenFactor >> 8);
            dst[odx + 2] = clamp(blueFactor >> 8);
            dst[odx + 3] = 255;

            redFactor = 298 * c1 + 409 * e + 128;
            greenFactor = 298 * c1 - 100 * d - 409 * e + 128;
            blueFactor = 298 * c1 + 516 * d + 128;

            dst[odx + 4] = clamp(redFactor >> 8);
            dst[odx + 5] = clamp(greenFactor >> 8);
            dst[odx + 6] = clamp(blueFactor >> 8);
            dst[odx + 7] = 255;

            #undef clamp
        }
    }
}