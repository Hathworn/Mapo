#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_unpack_yuy2_rgb8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < superPixCount) {
        int idx = i * 4;

        uint8_t y0 = src[idx];
        uint8_t u0 = src[idx + 1];
        uint8_t y1 = src[idx + 2];
        uint8_t v0 = src[idx + 3];

        int16_t c = y0 - 16;
        int16_t d = u0 - 128;
        int16_t e = v0 - 128;

        int t;
        #define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

        int odx = i * 6;

        // Precompute common expressions to optimize performance
        int c298 = 298 * c;
        int e409 = 409 * e + 128;
        
        dst[odx] = clamp((c298 +  e409) >> 8);
        dst[odx + 1] = clamp((c298 - 100 * d - e409) >> 8);
        dst[odx + 2] = clamp((c298 + 516 * d + 128) >> 8);

        c = y1 - 16;
        c298 = 298 * c;

        dst[odx + 3] = clamp((c298 + e409) >> 8);
        dst[odx + 4] = clamp((c298 - 100 * d - e409) >> 8);
        dst[odx + 5] = clamp((c298 + 516 * d + 128) >> 8);

        #undef clamp
    }
}