#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_unpack_yuy2_y16_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use loop unrolling to improve performance
    int idx;
    for (; i + 4 < superPixCount; i += stride * 4) {
        idx = i * 4;
        dst[idx] = 0;
        dst[idx + 1] = src[idx];
        dst[idx + 2] = 0;
        dst[idx + 3] = src[idx + 2];

        idx += stride * 4;
        dst[idx] = 0;
        dst[idx + 1] = src[idx];
        dst[idx + 2] = 0;
        dst[idx + 3] = src[idx + 2];

        idx += stride * 4;
        dst[idx] = 0;
        dst[idx + 1] = src[idx];
        dst[idx + 2] = 0;
        dst[idx + 3] = src[idx + 2];

        idx += stride * 4;
        dst[idx] = 0;
        dst[idx + 1] = src[idx];
        dst[idx + 2] = 0;
        dst[idx + 3] = src[idx + 2];
    }

    // Handle remaining iterations
    for (; i < superPixCount; i += stride) {
        idx = i * 4;
        dst[idx] = 0;
        dst[idx + 1] = src[idx];
        dst[idx + 2] = 0;
        dst[idx + 3] = src[idx + 2];
    }
}