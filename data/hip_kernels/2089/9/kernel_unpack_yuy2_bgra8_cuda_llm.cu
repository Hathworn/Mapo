#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_unpack_yuy2_bgra8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Ensure threads do not operate outside the bounds
    for (; i < superPixCount; i += stride) {
        int idx = i * 4;
        uint8_t y0 = src[idx];
        uint8_t u0 = src[idx + 1];
        uint8_t y1 = src[idx + 2];
        uint8_t v0 = src[idx + 3];

        // Precompute common constants
        int16_t d = u0 - 128;
        int16_t e = v0 - 128;
        
        // Use the inline function to avoid macro overhead
        auto clamp = [&](int x) { return (x > 255) ? 255 : (x < 0) ? 0 : x; };

        int odx = i * 8;

        // Compute common factors outside clamp function
        int16_t c0 = y0 - 16;
        dst[odx + 3] = 255;
        dst[odx + 2] = clamp((298 * c0 + 409 * e + 128) >> 8);
        dst[odx + 1] = clamp((298 * c0 - 100 * d - 409 * e + 128) >> 8);
        dst[odx] = clamp((298 * c0 + 516 * d + 128) >> 8);

        int16_t c1 = y1 - 16;
        dst[odx + 7] = 255;
        dst[odx + 6] = clamp((298 * c1 + 409 * e + 128) >> 8);
        dst[odx + 5] = clamp((298 * c1 - 100 * d - 409 * e + 128) >> 8);
        dst[odx + 4] = clamp((298 * c1 + 516 * d + 128) >> 8);
    }
}