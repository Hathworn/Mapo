#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_unpack_yuy2_bgr8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory to reduce memory access latency
    extern __shared__ uint8_t shared_src[];

    if (i < superPixCount) {
        int idx = i * 4;

        // Load input data into shared memory
        shared_src[threadIdx.x * 4] = src[idx];
        shared_src[threadIdx.x * 4 + 1] = src[idx + 1];
        shared_src[threadIdx.x * 4 + 2] = src[idx + 2];
        shared_src[threadIdx.x * 4 + 3] = src[idx + 3];
        __syncthreads(); // Synchronize threads within the block

        uint8_t y0 = shared_src[threadIdx.x * 4];
        uint8_t u0 = shared_src[threadIdx.x * 4 + 1];
        uint8_t y1 = shared_src[threadIdx.x * 4 + 2];
        uint8_t v0 = shared_src[threadIdx.x * 4 + 3];

        int16_t c = y0 - 16;
        int16_t d = u0 - 128;
        int16_t e = v0 - 128;

        int32_t t;
        #define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

        int odx = i * 6;

        dst[odx + 2] = clamp((298 * c + 409 * e + 128) >> 8);
        dst[odx + 1] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
        dst[odx] = clamp((298 * c + 516 * d + 128) >> 8);

        c = y1 - 16;

        dst[odx + 5] = clamp((298 * c + 409 * e + 128) >> 8);
        dst[odx + 4] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
        dst[odx + 3] = clamp((298 * c + 516 * d + 128) >> 8);

        #undef clamp
    }
}