#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ bool between(int value, int lowerBound, int upperBound)
{
    return (value >= lowerBound && value <= upperBound);
}

__device__ void getTopLeft(float x, int width, int& point, float& weight)
{
    float xcoord = (x + 1) * (width - 1) / 2;
    point = floor(xcoord);
    weight = 1 - (xcoord - point);
}

__global__ void bilinearSamplingFromGrid(float* inputImages_data, int inputImages_strideBatch, int inputImages_strideChannels, int inputImages_strideHeight, int inputImages_strideWidth, float* grids_data, int grids_strideBatch, int grids_strideYX, int grids_strideHeight, int grids_strideWidth, float* output_data, int output_strideBatch, int output_strideChannels, int output_strideHeight, int output_strideWidth, int inputImages_channels, int inputImages_height, int inputImages_width, int output_width)
{
    const int xOut = blockIdx.x * blockDim.y + threadIdx.y;
    const bool withinImageBounds = xOut < output_width;
    const bool withinGridBounds = blockIdx.x * blockDim.y + threadIdx.x / 2 < output_width;
    const int yOut = blockIdx.y;
    const int width = inputImages_width;
    const int height = inputImages_height;
    const int b = blockIdx.z;

    float yf, xf;

    __shared__ float gridData[32];
    
    if (threadIdx.y == 0 && withinGridBounds) {
        gridData[threadIdx.x] = grids_data[b * grids_strideBatch + yOut * grids_strideHeight + xOut * grids_strideWidth + threadIdx.x];
    }
    
    // Synchronize threads to ensure all grid data is loaded
    __syncthreads();
    
    if (!withinImageBounds) return;

    // Load grid coordinates
    yf = gridData[threadIdx.y * 2];
    xf = gridData[threadIdx.y * 2 + 1];

    int yInTopLeft, xInTopLeft;
    float yWeightTopLeft, xWeightTopLeft;
    getTopLeft(xf, inputImages_width, xInTopLeft, xWeightTopLeft);
    getTopLeft(yf, inputImages_height, yInTopLeft, yWeightTopLeft);

    const int outAddress = output_strideBatch * b + output_strideHeight * yOut + output_strideWidth * xOut;

    // Calculate base input addresses
    const int inTopLeftAddress = inputImages_strideBatch * b + inputImages_strideHeight * yInTopLeft + inputImages_strideWidth * xInTopLeft;
    const int inTopRightAddress = inTopLeftAddress + inputImages_strideWidth;
    const int inBottomLeftAddress = inTopLeftAddress + inputImages_strideHeight;
    const int inBottomRightAddress = inBottomLeftAddress + inputImages_strideWidth;

    // Pre-compute boolean checks for boundary conditions
    bool topLeftIsIn = between(xInTopLeft, 0, width - 1) && between(yInTopLeft, 0, height - 1);
    bool topRightIsIn = between(xInTopLeft + 1, 0, width - 1) && between(yInTopLeft, 0, height - 1);
    bool bottomLeftIsIn = between(xInTopLeft, 0, width - 1) && between(yInTopLeft + 1, 0, height - 1);
    bool bottomRightIsIn = between(xInTopLeft + 1, 0, width - 1) && between(yInTopLeft + 1, 0, height - 1);

    // Interpolation
    for (int t = threadIdx.x; t < inputImages_channels; t += blockDim.x) {
        float v = 0.0f;
        float inTopLeft = topLeftIsIn ? inputImages_data[inTopLeftAddress + t] : 0.0f;
        float inTopRight = topRightIsIn ? inputImages_data[inTopRightAddress + t] : 0.0f;
        float inBottomLeft = bottomLeftIsIn ? inputImages_data[inBottomLeftAddress + t] : 0.0f;
        float inBottomRight = bottomRightIsIn ? inputImages_data[inBottomRightAddress + t] : 0.0f;

        // Weighted interpolation
        v = xWeightTopLeft * yWeightTopLeft * inTopLeft
          + (1 - xWeightTopLeft) * yWeightTopLeft * inTopRight
          + xWeightTopLeft * (1 - yWeightTopLeft) * inBottomLeft
          + (1 - xWeightTopLeft) * (1 - yWeightTopLeft) * inBottomRight;

        output_data[outAddress + t] = v;
    }
}