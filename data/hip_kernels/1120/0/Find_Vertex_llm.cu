#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define V 8
#define E 11
#define MAX_WEIGHT 1000000
#define TRUE    1
#define FALSE   0

typedef int boolean;

typedef struct {
    int u;
    int v;
} Edge;

typedef struct {
    int title;
    boolean visited;
} Vertex;

__device__ __host__ int findEdge(Vertex u, Vertex v, Edge *edges, int *weights) {
    for(int i = 0; i < E; i++) {
        if(edges[i].u == u.title && edges[i].v == v.title) {
            return weights[i];
        }
    }
    return MAX_WEIGHT;
}

__global__ void Find_Vertex(Vertex *vertices, Edge *edges, int *weights, int *length, int *updateLength) {
    int u = blockIdx.x * blockDim.x + threadIdx.x;  // Optimize thread index calculation

    if (u < V && vertices[u].visited == FALSE) {    // Check if u is within bounds
        vertices[u].visited = TRUE;

        for (int v = 0; v < V; v++) {
            int weight = findEdge(vertices[u], vertices[v], edges, weights);

            if (weight < MAX_WEIGHT) {
                atomicMin(&updateLength[v], length[u] + weight);  // Use atomicMin for thread safety
            }
        }
    }
}