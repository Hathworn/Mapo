#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define V 8
#define E 11
#define MAX_WEIGHT 1000000
#define TRUE    1
#define FALSE   0

typedef int boolean;

// Represents an edge or path between Vertices
typedef struct
{
    int u;
    int v;
} Edge;

// Represents a Vertex
typedef struct
{
    int title;
    boolean visited;
} Vertex;

// Updates the paths for each vertex
__global__ void Update_Paths(Vertex *vertices, int *length, int *updateLength)
{
    int u = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate unique thread index
    if (u < V)  // Ensure we don't exceed the number of vertices
    {
        if (length[u] > updateLength[u])
        {
            length[u] = updateLength[u];
            vertices[u].visited = FALSE;
        }
        updateLength[u] = length[u];
    }
}