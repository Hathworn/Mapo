#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil_sync(int *in, int *out)
{
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x + RADIUS;

    // Load elements with boundary checks into shared memory
    if (gindex < gridDim.x * blockDim.x + 2 * RADIUS) {
        temp[lindex] = in[gindex + RADIUS];
    }
    if (threadIdx.x < RADIUS) {
        int left_index = gindex - RADIUS;
        int right_index = gindex + BLOCK_SIZE + RADIUS;
        temp[lindex - RADIUS] = left_index >= 0 ? in[left_index] : 0; // Boundary check
        temp[lindex + BLOCK_SIZE] = right_index < gridDim.x * blockDim.x + 2 * RADIUS ? in[right_index] : 0; // Boundary check
    }

    // Synchronize threads before applying stencil
    __syncthreads();

    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS; offset <= RADIUS; offset++)
        result += temp[lindex + offset];

    // Store the result
    out[gindex] = result;
}