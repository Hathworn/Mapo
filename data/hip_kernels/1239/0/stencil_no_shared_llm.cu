#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 1024

#ifndef RADIUS
#define RADIUS 3
#endif

#ifndef ITERS
#define ITERS 100
#endif

#ifndef USE_L2
#define USE_L2 false
#endif

__global__ void stencil_no_shared(int *in, int *out)
{
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int result = 0;

    // Loop unrolling for efficiency
    for (int offset = -RADIUS; offset <= RADIUS; offset += 2) {
        result += in[gindex + offset] + in[gindex + offset + 1];
    }
    
    // Store the result
    out[gindex] = result;
}