#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 1024

#ifndef RADIUS
#define RADIUS 3
#endif

#ifndef ITERS
#define ITERS 100
#endif

#ifndef USE_L2
#define USE_L2 false
#endif

__global__ void stencil_shared(int *in, int *out)
{
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x + RADIUS;

    // Optimize: Use shared memory efficiently for boundary conditions
    temp[lindex] = in[gindex + RADIUS];
    if (threadIdx.x < RADIUS) {
        temp[lindex - RADIUS] = in[gindex];
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE + RADIUS];
    }
    __syncthreads();

    // Optimize: Use loop unrolling for performance
    int result = temp[lindex - 3] + temp[lindex - 2] + temp[lindex - 1] 
                 + temp[lindex] 
                 + temp[lindex + 1] + temp[lindex + 2] + temp[lindex + 3];

    // Store the result
    out[gindex] = result;
}