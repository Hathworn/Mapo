#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateVelocity_k(float2 *v, float *vx, float *vy, int dx, int pdx, int dy, int lb, size_t pitch) {
    
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    float scale = 1.f / (dx * dy); // Pre-compute scale factor outside the loop for efficiency

    // gtidx is the domain location in x for this thread
    if (gtidx < dx) {
        for (int p = 0; p < lb; p++) {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;
            if (fi < dy) {
                int fjr = fi * pdx + gtidx;
                float2 nvterm; // Move variable declaration here to limit scope
                nvterm.x = vx[fjr] * scale;
                nvterm.y = vy[fjr] * scale;

                float2 *fj = (float2*)((char*)v + fi * pitch) + gtidx;
                *fj = nvterm;
            }
        }
    }
}