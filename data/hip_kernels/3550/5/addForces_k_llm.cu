#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addForces_k(float2 *v, int dx, int dy, int spx, int spy, float fx, float fy, int r, size_t pitch) {

    // Cache thread indices to enhance readability
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // Compute the correct memory address with pitched memory access
    float2 *fj = (float2*)((char*)v + (ty + spy) * pitch) + tx + spx;

    // Load memory to register for fast access
    float2 vterm = *fj;
    tx -= r; 
    ty -= r;
    
    // Reduce redundant computations by precalculating squares
    float txx2 = tx * tx;
    float tyy2 = ty * ty;
    float s = 1.f / (1.f + txx2 * txx2 + tyy2 * tyy2);
    
    // Update values using shared memory for faster read and write (if applicable)
    vterm.x += s * fx;
    vterm.y += s * fy;
    
    // Store the result back to the global memory
    *fj = vterm;
}