#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int local_factor = *factor; // Load factor into register to reduce global memory access

    for (int i = 0; i < num_iterations; i++)
    {
        g_data[idx] += local_factor; // Use local register for repeated access
    }
}