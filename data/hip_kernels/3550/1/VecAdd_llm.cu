#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int N)
{
    // Calculate unique thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Use conditional operator for improved performance
    C[i] = (i < N) ? (A[i] + B[i]) : C[i];
}