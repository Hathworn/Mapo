#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(int *array, int arrayCount)
{
    extern __shared__ int dynamicSmem[];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use vectorized load/store and avoid redundant memory access when possible
    for (int i = idx; i < arrayCount; i += blockDim.x * gridDim.x) {
        array[i] *= array[i];
    }
}