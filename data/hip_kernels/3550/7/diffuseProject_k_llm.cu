#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diffuseProject_k(float2 *vx, float2 *vy, int dx, int dy, float dt, float visc, int lb) {
    // Calculate global thread indices
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Return early if gtidx is out of bounds
    if (gtidx >= dx) return;

    for (int p = 0; p < lb; p++) {
        // Compute global y-index with batching
        int fi = gtidy * lb + p;
        
        if (fi < dy) {
            int fj = fi * dx + gtidx;
            float2 xterm = vx[fj];
            float2 yterm = vy[fj];
            
            // Calculate wave numbers
            int iix = gtidx;
            int iiy = (fi > dy / 2) ? (fi - dy) : fi;
            
            // Velocity diffusion
            float kk = static_cast<float>(iix * iix + iiy * iiy);
            float diff = 1.f / (1.f + visc * dt * kk);
            xterm.x *= diff; xterm.y *= diff;
            yterm.x *= diff; yterm.y *= diff;

            // Velocity projection
            if (kk > 0.f) {
                float rkk = 1.f / kk;
                float rkp = (iix * xterm.x + iiy * yterm.x);
                float ikp = (iix * xterm.y + iiy * yterm.y);
                
                // Apply projection results
                xterm.x -= rkk * rkp * iix;
                xterm.y -= rkk * ikp * iix;
                yterm.x -= rkk * rkp * iiy;
                yterm.y -= rkk * ikp * iiy;
            }
            
            // Store updated terms back to global memory
            vx[fj] = xterm;
            vy[fj] = yterm;
        }
    }
}