#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void advectParticles_OGL(float2 *part, float2 *v, int dx, int dy, float dt, int lb, size_t pitch) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    // gtidx is the domain location in x for this thread
    if (gtidx < dx) {
        for (p = 0; p < lb; p++) {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;
            if (fi < dy) {
                int fj = fi * dx + gtidx;
                float2 pterm = part[fj];
                
                int xvi = min(max((int)(pterm.x * dx), 0), dx - 1); // Improved bounds check
                int yvi = min(max((int)(pterm.y * dy), 0), dy - 1); // Improved bounds check
                
                float2 vterm = *((float2*)((char*)v + yvi * pitch) + xvi);
                
                // Streamline position update using fmodf to handle wrap-around
                pterm.x = fmodf(pterm.x + dt * vterm.x + 1.f, 1.f);
                pterm.y = fmodf(pterm.y + dt * vterm.y + 1.f, 1.f);

                part[fj] = pterm;
            }
        } // If this thread is inside the domain in Y
    } // If this thread is inside the domain in X
}