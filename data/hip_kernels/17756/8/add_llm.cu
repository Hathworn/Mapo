#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(float* vec_a, float* vec_b, float* vec_c, int n)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop to handle large arrays efficiently
    int stride = blockDim.x * gridDim.x;
    for (; i < n; i += stride)
    {
        vec_c[i] = vec_a[i] + vec_b[i];
    }
}