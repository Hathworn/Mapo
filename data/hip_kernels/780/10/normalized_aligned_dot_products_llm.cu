#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalized_aligned_dot_products(const double* __restrict__ A, const double divisor, const unsigned int m, const unsigned int n, double* __restrict__ QT)
{
    // Use shared memory if multiple threads in block need the same data
    int a = blockIdx.x * blockDim.x + threadIdx.x;

    // Only compute this when within bounds
    if (a < n) {
        QT[a] = A[a + m - 1] / divisor;
    }
}