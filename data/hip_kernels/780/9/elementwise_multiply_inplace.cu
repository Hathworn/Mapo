#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementwise_multiply_inplace(const hipDoubleComplex* A, hipDoubleComplex* B, const int size)
{
int tid = blockIdx.x * blockDim.x + threadIdx.x;
if(tid < size) {
B[tid] = hipCmul(A[tid], B[tid]);
}
}