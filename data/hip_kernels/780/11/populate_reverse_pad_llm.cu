#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void populate_reverse_pad(const double *Q, double *Q_reverse_pad, const double *mean, const int window_size, const int size)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    double mu = *mean;

    // Use ternary operator for concise condition check
    Q_reverse_pad[tid] = (tid < window_size) ? (Q[window_size - 1 - tid] - mu) : ((tid < size) ? 0 : Q_reverse_pad[tid]);
}