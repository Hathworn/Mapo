#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define FIRST_RUN 0
const float WL  = 10.0;
const float XL1 = -WL;
const float XL2 =  WL;
const float YL1 = -WL;
const float YL2 =  WL;

const float rsrc = 0.1;
const float ldc  = 0.5;
const float xsrc = 0.0;
const float ysrc = 0.0;

__global__ void ray_shoot(int *maxX, int *maxY, float *lens_scale, float *xlens, float *ylens, float*eps, int *num_lenses, float *dev_arr)
{
    // Calculate the unique index for each thread
    int threadBlockPos = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Determine the y and x position
    int y = threadBlockPos / (*maxY);
    int x = threadBlockPos - ((*maxX) * y);

    const float rsrc2 = rsrc * rsrc;

    // Calculate the lens plane coordinates
    float xl = XL1 + x * (*lens_scale);
    float yl = YL1 + y * (*lens_scale);

    float xs = xl;
    float ys = yl;

    float dx, dy, dr;
    
    // Loop over lenses to calculate image positions
    for(int p = 0; p < (*num_lenses); ++p){
        dx = xl - xlens[p];
        dy = yl - ylens[p];
        dr = dx * dx + dy * dy;
        
        // Avoid potential division by zero
        if (dr != 0.0f) {
            xs -= eps[p] * dx / dr;
            ys -= eps[p] * dy / dr;
        }
    }

    // Calculate separation squared from the source
    float xd = xs - xsrc;
    float yd = ys - ysrc;
    float sep2 = (xd * xd) + (yd * yd);

    // Check if within the source radius
    if(sep2 < rsrc2){
        float mu = sqrtf(1.0f-sep2/rsrc2);
        dev_arr[threadBlockPos] = 1.0 - ldc * (1-mu);
    }
}