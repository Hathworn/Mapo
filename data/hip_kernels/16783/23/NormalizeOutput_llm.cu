#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Using a simplified loop and reducing global memory accesses
    if (idx < num_elements) {
        int64_t normalized_val = static_cast<int64_t>(original[idx]);
        to_normalize[idx * 3] = batch_index;
        to_normalize[idx * 3 + 1] = class_index;
        to_normalize[idx * 3 + 2] = normalized_val;
    }
}