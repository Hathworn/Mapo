#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Use thread and block indices to initialize different states
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, index, 0, &state[index]);
}