#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_product(float *a, float *b, float *c, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global index
    if (idx < n) { // Ensure index is within array bounds
        c[idx] = a[idx] * b[idx]; // Perform dot product at global index
    }
}