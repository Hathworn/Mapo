#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Calculate blockDim-based unique thread index in the grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Perform division only if unique thread index is zero
    if (idx == 0) {
        *accuracy /= N;
    }
}