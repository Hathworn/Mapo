#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelA(float* r, float* x, float* y, float* z, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int stride = blockDim.x * gridDim.x; // Calculate stride for the grid
    for (int i = idx; i < size; i += stride) { // Loop with global stride
        r[i] = x[i] * y[i] + z[i];
    }
}