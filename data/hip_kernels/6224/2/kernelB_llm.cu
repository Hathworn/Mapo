#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int stride = blockDim.x * gridDim.x; // Calculate stride for each thread
    for (int i = index; i < size; i += stride) {
        r[i] = x[i] * y[i] + z[i]; // Perform vector operation
    }
}