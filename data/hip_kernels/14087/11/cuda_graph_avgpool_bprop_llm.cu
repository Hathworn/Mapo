#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_graph_avgpool_bprop(float* gradInput, const float *gradOutput, const float* clusters, const int nClusters, const int poolsize, const int dim, const int nClustersPerThread) {

    extern __shared__ float shared_mem[];
    float* gradOutput_data = (float*)shared_mem;

    const int tidx = threadIdx.x;
    gradInput += blockIdx.x * dim;
    gradOutput += blockIdx.x * nClusters;

    // Load gradOutput into shared memory for faster access
    for (int i = tidx; i < nClusters; i += blockDim.x) {
        gradOutput_data[i] = gradOutput[i];
    }
    __syncthreads();

    // Distribute gradOutput over gradInput using clusters
    for (int i = tidx; i < poolsize; i += blockDim.x) {
        for (int j = 0; j < nClusters; ++j) {
            atomicAdd(&gradInput[(int)(clusters[j * poolsize + i] - 1)], gradOutput_data[j] / poolsize);
        }
    }
}