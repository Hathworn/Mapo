#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void batch_crop_kernel(float* input, const int nCropRows, const int nCropCols, const int iH, const int iW, const int nPlanes) {
    const int plane = blockIdx.x;
    if (plane >= nPlanes) return;

    input += plane * iH * iW;
    const int tx = threadIdx.x + blockIdx.y * blockDim.x; // Flatten x-dimension
    const int ty = threadIdx.y + blockIdx.z * blockDim.y; // Flatten y-dimension

    // Ensure threads are within bounds and check crop conditions
    if (ty < iH && (ty >= iH - nCropRows || ty < nCropRows) && tx < iW) {
        input[ty * iW + tx] = 0;
    }
    if (tx < iW && (tx >= iW - nCropCols || tx < nCropCols) && ty < iH) {
        input[ty * iW + tx] = 0;
    }
}