#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void modulus_updateGradInput_kernel(float* input, float* output, float* gradInput, float* gradOutput, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Thread boundary condition
    if (i < n) {
        const float eps = 0.0001;
        const float c = gradOutput[i] / max(output[i], eps);
        
        // Vectorized memory access
        gradInput[i * 2] = input[i * 2] * c;
        gradInput[i * 2 + 1] = input[i * 2 + 1] * c;
    }
}