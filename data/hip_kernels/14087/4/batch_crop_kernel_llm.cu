#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void batch_crop_kernel(float* input, const int nCropRows, const int nCropCols, const int iH, const int iW, const int nPlanes) {
    const int plane = blockIdx.x;
    if (plane >= nPlanes) return;

    input += plane * iH * iW;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int tz = threadIdx.z;

    // Simplified boundary checks using a single conditional
    if (tz < 4) {
        int index;
        switch (tz) {
            case 0: // top
                index = ty * iW + tx;
                break;
            case 1: // bottom
                index = (iH - ty - 1) * iW + tx;
                break;
            case 2: // left
                index = tx * iW + ty;
                break;
            case 3: // right
                index = tx * iW + (iW - ty - 1);
                break;
        }
        input[index] = 0;
    }
}