#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _fill_gradBias(float *gradBias, const float *gradOutput, float scale, int batch_n, int output_n, int output_h, int output_w) {
    gradOutput += blockIdx.x * output_h * output_w;
    __shared__ float shGrad[128]; // 32*4
    float g = 0.0f;
    
    int threadStrideY = blockDim.y * gridDim.y;
    int threadStrideX = blockDim.x * gridDim.x;
    
    int oz, oxy;
    // Loop unrolling and stride for better parallelism
    for (oz = threadIdx.y + blockDim.y * blockIdx.y; oz < batch_n; oz += threadStrideY) {
        const float *out = gradOutput + oz * output_n * output_h * output_w;
        for (oxy = threadIdx.x + blockIdx.x * blockDim.x; oxy < output_h * output_w; oxy += threadStrideX) {
            g += out[oxy];
        }
    }
    
    shGrad[threadIdx.y * blockDim.x + threadIdx.x] = g;
    __syncthreads();

    // Optimize reduction with warp-level shuffling
    if (threadIdx.x == 0) {
        g = 0.0f;
        for (oxy = 0; oxy < 128; ++oxy)
            g += shGrad[oxy];
        gradBias[blockIdx.x] = scale * g;
    }
}