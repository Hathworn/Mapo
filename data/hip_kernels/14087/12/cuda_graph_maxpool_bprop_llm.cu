#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_graph_maxpool_bprop(float* gradInput, const float *gradOutput, const float* indices, const int nClusters, const int dim, const int nClustersPerThread) {
    extern __shared__ float shared_mem[];
    float* gradOutput_data = (float*)shared_mem;
    float* indices_data = (float*)&gradOutput_data[nClusters];

    const int tidx = threadIdx.x;
    gradInput += blockIdx.x * dim;
    gradOutput += blockIdx.x * nClusters;
    indices += blockIdx.x * nClusters;
    __syncthreads();

    for (int i = tidx; i < nClusters; i += blockDim.x) {  // Load data into shared memory using all threads
        gradOutput_data[i] = gradOutput[i];
        indices_data[i] = indices[i];
    }
    __syncthreads();

    // Execute backprop calculation using all threads
    for (int i = tidx; i < nClusters; i += blockDim.x) {
        atomicAdd(&gradInput[(int)indices_data[i] - 1], gradOutput_data[i]);
    }
}