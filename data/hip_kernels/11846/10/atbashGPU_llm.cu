#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void atbashGPU(char const *in, char *out, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        // Simplified the reversal logic for clearer intention
        int targetIdx = n - 1 - i; 
        out[targetIdx] = in[i];
    }
}