#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpuMatMul(float *a, float *b, float *c, int m, int n, int p) {
    // Calculate row index of the element
    uint row = blockDim.y * blockIdx.y + threadIdx.y;
    // Calculate column index of the element
    uint col = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if each thread needs to proceed with the calculations
    if (row < m && col < p) {
        float val = 0.0f;

        // Use shared memory for tiling to improve data reuse
        __shared__ float shared_a[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float shared_b[BLOCK_SIZE][BLOCK_SIZE];
        
        for (int k = 0; k < (n + BLOCK_SIZE - 1) / BLOCK_SIZE; ++k) {
            if (k * BLOCK_SIZE + threadIdx.x < n && row < m) {
                shared_a[threadIdx.y][threadIdx.x] = a[row * n + k * BLOCK_SIZE + threadIdx.x];
            } else {
                shared_a[threadIdx.y][threadIdx.x] = 0.0f;
            }

            if (k * BLOCK_SIZE + threadIdx.y < n && col < p) {
                shared_b[threadIdx.y][threadIdx.x] = b[(k * BLOCK_SIZE + threadIdx.y) * p + col];
            } else {
                shared_b[threadIdx.y][threadIdx.x] = 0.0f;
            }

            // Synchronize to ensure all data is loaded before computation
            __syncthreads();
            
            // Compute partial product for this block and add to accumulator
            for (int e = 0; e < BLOCK_SIZE; ++e) {
                val += shared_a[threadIdx.y][e] * shared_b[e][threadIdx.x];
            }

            // Synchronize to ensure all computations finished before loading next tile
            __syncthreads();
        }
        
        // Store the result
        c[row * p + col] = val;
    }
}
```
