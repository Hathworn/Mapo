#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello(char *a, int *b) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Compute global index
    a[idx] += b[idx]; // Use global index for accessing arrays
}