#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpuTranspose(float *a, float *b, int m, int n) {
    // Utilize shared memory for coalesced memory access
    __shared__ float tile[32][32 + 1]; // Padding to avoid bank conflicts

    uint xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    uint yIndex = blockDim.y * blockIdx.y + threadIdx.y;

    if (xIndex < n && yIndex < m) {
        // Load from global memory to shared memory
        tile[threadIdx.y][threadIdx.x] = a[yIndex * n + xIndex];
    }

    __syncthreads();

    xIndex = blockDim.x * blockIdx.y + threadIdx.x; // Transpose block offset
    yIndex = blockDim.y * blockIdx.x + threadIdx.y;

    if (xIndex < m && yIndex < n) {
        // Write from shared memory to global memory
        b[yIndex * m + xIndex] = tile[threadIdx.x][threadIdx.y];
    }
}