#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NmDistanceGradKernel(int b, int n, const float *xyz1, int m, const float *xyz2, const float *grad_dist1, const int *idx1, float *grad_xyz1, float *grad_xyz2) {
    int batchIdx = blockIdx.x;
    int threadIdxGlobal = threadIdx.x + blockIdx.y * blockDim.x;
    
    // Iterate over batches
    for (; batchIdx < b; batchIdx += gridDim.x) {
        // Iterate over elements within a batch
        for (int j = threadIdxGlobal; j < n; j += blockDim.x * gridDim.y) {
            float x1 = xyz1[(batchIdx * n + j) * 2 + 0];
            float y1 = xyz1[(batchIdx * n + j) * 2 + 1];
            int j2 = idx1[batchIdx * n + j];
            float x2 = xyz2[(batchIdx * m + j2) * 2 + 0];
            float y2 = xyz2[(batchIdx * m + j2) * 2 + 1];
            float gradient = grad_dist1[batchIdx * n + j] * 2;
            
            // Compute gradients
            atomicAdd(&(grad_xyz1[(batchIdx * n + j) * 2 + 0]), gradient * (x1 - x2));
            atomicAdd(&(grad_xyz1[(batchIdx * n + j) * 2 + 1]), gradient * (y1 - y2));
            atomicAdd(&(grad_xyz2[(batchIdx * m + j2) * 2 + 0]), -(gradient * (x1 - x2)));
            atomicAdd(&(grad_xyz2[(batchIdx * m + j2) * 2 + 1]), -(gradient * (y1 - y2)));
        }
    }
}