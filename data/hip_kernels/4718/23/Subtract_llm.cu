#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Calculate global flattened index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;

    // Check within array bounds
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}