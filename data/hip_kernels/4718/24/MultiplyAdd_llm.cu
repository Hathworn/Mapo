#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Calculate global coordinates using block and thread indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;

    // Perform computation only within valid range
    if (x < width && y < height)
    {
        d_Result[p] = __fmul_rn(d_ConstantA[0], d_Data[p]) + d_ConstantB[0]; // Efficient fused multiply-add
    }
}