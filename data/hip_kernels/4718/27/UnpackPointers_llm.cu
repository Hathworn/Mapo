#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;

    // Optimize loop bounds to cover entire image height
    for (int y = 0; y < h; y += 32) {
        for (int x = 0; x < w; x += 16) {
            unsigned int val = minmax[(y / 32) * w + x + tx];
            if (val) {
                for (int k = 0; k < 32 && numPts < maxPts; k++) {
                    if (val & 0x1) {
                        ptrs[16 * numPts + tx] = (y + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}