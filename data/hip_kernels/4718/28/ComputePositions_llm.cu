#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputePositions(float *g_Data1, float *g_Data2, float *g_Data3, int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
    int i = blockIdx.x * POSBLK_SIZE + threadIdx.x;  // Simplified index calculation
    if (i >= numPts) return;
    
    int p = d_Ptrs[i];
    
    // Load data into shared for coalesced global memory access
    float val[] = { g_Data2[p], g_Data2[p-1], g_Data2[p+1], g_Data2[p-w], g_Data2[p+w], g_Data3[p], g_Data1[p] };

    float dx = 0.5f * (val[2] - val[1]);
    float dxx = 2.0f * val[0] - val[1] - val[2];
    
    float dy = 0.5f * (val[4] - val[3]);
    float dyy = 2.0f * val[0] - val[3] - val[4];
    
    float ds = 0.5f * (val[6] - val[5]);
    float dss = 2.0f * val[0] - val[5] - val[6];
    
    float dxy = 0.25f * (g_Data2[p+w+1] + g_Data2[p-w-1] - g_Data2[p-w+1] - g_Data2[p+w-1]);
    float dxs = 0.25f * (g_Data3[p+1] + g_Data1[p-1] - g_Data1[p+1] - g_Data3[p-1]);
    float dys = 0.25f * (g_Data3[p+w] + g_Data1[p-w] - g_Data3[p-w] - g_Data1[p+w]);
    
    // Precompute reuse variables to reduce calculation redundancy
    float dxx_dyy = dxx * dyy;
    float dss_dyy = dss * dyy;
    float dss_dxx = dss * dxx;
    float dxy_dxy = dxy * dxy;
    float dxs_dxs = dxs * dxs;
    float dys_dys = dys * dys;
    float dys_dxs = dys * dxs;
    float idxx = dss_dyy - dys_dys;
    float idxy = dys_dxs - dxy * dss;
    float idxs = dxy * dys - dyy * dxs;
    float idet = 1.0f / (idxx * dxx + idxy * dxy + idxs * dxs);

    // Calculate shifts
    float pdx = idet * (idxx * dx + idxy * dy + idxs * ds);
    float pdy = idet * (idxy * dx + (dxx_dyy - dxs_dxs) * dy + dxy * ds - dyy * dxs);
    float pds = idet * (idxs * dx + (dxy * dxs - dxx * dys) * dy + (dxx_dyy - dxy_dxy) * ds);

    // Bound check for correct localization
    if (pdx < -0.5f || pdx > 0.5f || pdy < -0.5f || pdy > 0.5f || pds < -0.5f || pds > 0.5f)
    {
        pdx = __fdividef(dx, dxx);
        pdy = __fdividef(dy, dyy);
        pds = __fdividef(ds, dss);
    }
    
    float dval = 0.5f * (dx * pdx + dy * pdy + ds * pds);

    // Write results into output array
    d_Sift[i + 0 * maxPts] = (p % w) + pdx;
    d_Sift[i + 1 * maxPts] = (p / w) + pdy;
    d_Sift[i + 2 * maxPts] = d_ConstantA[0] * exp2f(pds * d_ConstantB[0]);
    d_Sift[i + 3 * maxPts] = val[0] + dval;
    
    float tra = dxx + dyy;
    float det = dxx_dyy - dxy_dxy;
    d_Sift[i + 4 * maxPts] = __fdividef(tra * tra, det);
}