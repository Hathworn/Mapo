#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;  // Optimize with blockDim for flexibility
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int sz = height * pitch;
    
    // Removed p calculation inside bounds check
    if (x < width && y < height) 
    {
        int p = threadIdx.z * sz + y * pitch + x;
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
    // Removed __syncthreads() as it's unnecessary here
}