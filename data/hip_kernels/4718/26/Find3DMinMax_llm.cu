#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int height) {
    // Data cache with reduced size using registers where applicable
    __shared__ float data1[3 * (MINMAX_SIZE + 2)];
    __shared__ float data2[3 * (MINMAX_SIZE + 2)];
    __shared__ float data3[3 * (MINMAX_SIZE + 2)];

    // Use local registers for ymin and ymax to reduce shared memory usage
    float ymin1, ymin2, ymin3, ymax1, ymax2, ymax3;

    const int tx = threadIdx.x;
    const int xStart = __mul24(blockIdx.x, MINMAX_SIZE);
    const int xEndClamped = min(xStart + MINMAX_SIZE - 1, width - 1);
    int memWid = MINMAX_SIZE + 2;

    int memPos0 = tx - WARP_SIZE + 1;
    int yq = 0;
    unsigned int output = 0;

    for (int y = 0; y < 34; y++) {
        output >>= 1;

        int yp = 32 * blockIdx.y + y - 1;
        yp = max(yp, 0);
        yp = min(yp, height - 1);
        int readStart = __mul24(yp, width);

        // Set the entire data cache contents with boundary check
        int xReadPos = xStart + tx - WARP_SIZE;
        if (tx >= (WARP_SIZE - 1)) {
            if (xReadPos < 0 || xReadPos >= width) {
                data1[memPos0] = data2[memPos0] = data3[memPos0] = 0.0f;
            } else {
                data1[memPos0] = d_Data1[readStart + xReadPos];
                data2[memPos0] = d_Data2[readStart + xReadPos];
                data3[memPos0] = d_Data3[readStart + xReadPos];
            }
        }
        __syncthreads();

        int memPos2 = yq * memWid + tx;
        if (y > 1 && tx < memWid) {
            // Calculate min and max using registers to reduce shared memory load
            ymin1 = fminf(fminf(data1[memPos0], data1[memPos0 + memWid]), data1[memPos2]);
            ymin2 = fminf(fminf(data2[memPos0], data2[memPos0 + memWid]), data2[memPos2]);
            ymin3 = fminf(fminf(data3[memPos0], data3[memPos0 + memWid]), data3[memPos2]);
            ymax1 = fmaxf(fmaxf(data1[memPos0], data1[memPos0 + memWid]), data1[memPos2]);
            ymax2 = fmaxf(fmaxf(data2[memPos0], data2[memPos0 + memWid]), data2[memPos2]);
            ymax3 = fmaxf(fmaxf(data3[memPos0], data3[memPos0 + memWid]), data3[memPos2]);
        }
        __syncthreads();

        if (y > 1 && tx < MINMAX_SIZE) {
            int xWritePos = xStart + tx;
            if (xWritePos <= xEndClamped) {
                // Consolidate min and max comparisons using local variables
                float minv = fminf(fminf(fminf(fminf(ymin2, ymin2 + 2), ymin1), ymin3), d_ConstantA[1]);
                float maxv = fmaxf(fmaxf(fmaxf(fmaxf(ymax2, ymax2 + 2), ymax1), ymax3), d_ConstantA[0]);
                if (data2[memPos0 + 1] < minv || data2[memPos0 + 1] > maxv) 
                    output |= 0x80000000;
            }
        }
        __syncthreads();

        memPos0 = yq * memWid + (tx - WARP_SIZE + 2);
        yq = (yq + 1) % 3;
    }

    if (tx < MINMAX_SIZE && (xStart + tx) < width) {
        int writeStart = __mul24(blockIdx.y, width) + xStart + tx;
        d_Result[writeStart] = output;
    }
}