#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];

    const int tx = threadIdx.x;
    const int x = blockIdx.x * blockDim.x + tx;
    const int y = blockIdx.y * blockDim.y;
    
    int p = y * pitch + x;

    // Load data into shared memory and handle out-of-bounds
    float val = (x < width) ? d_Data[p] : d_Data[p - tx];
    minvals[tx] = val;
    maxvals[tx] = val;

    // Loop through y-dimension within block and compute local min/max
    for (int ty = 1; ty < blockDim.y; ty++) {
        p += pitch;
        if (x < width) {
            val = d_Data[p];
            minvals[tx] = fminf(minvals[tx], val);
            maxvals[tx] = fmaxf(maxvals[tx], val);
        }
    }
    __syncthreads();

    // Reduce within block to find min and max
    for (int d = blockDim.x / 2; d > 0; d >>= 1) {
        if (tx < d) {
            minvals[tx] = fminf(minvals[tx], minvals[tx + d]);
            maxvals[tx] = fmaxf(maxvals[tx], maxvals[tx + d]);
        }
        __syncthreads();
    }

    // Write result to global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr + 0] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}