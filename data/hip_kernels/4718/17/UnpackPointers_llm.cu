#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    const int stride = 32 * w;
    int numPts = 0;

    for (int y = 0; y < h; y += 32) {
        for (int x = 0; x < w; x += 16) {
            unsigned int val = minmax[(y * w + x) / 32 + tx];
            if (val) {
                for (int k = 0; k < 32; ++k) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = (y + k) * w + x + tx; // Direct pointer calculation
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}