#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved function to reduce redundant memory lookups
__device__ float computeExp(float i2sigma2, int idx) {
    return exp(i2sigma2 * (idx - 7) * (idx - 7));
}

__device__ int boundIndex(int idx, int bound) {
    if (idx < 0) return 0;
    if (idx >= bound) return bound - 1;
    return idx;
}

__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h) {
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];

    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Reset histograms
    for (int i = 0; i < 13; i++)
        hist[i * 32 + tx] = 0.0f;
    __syncthreads();

    // Precompute Gaussian values
    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    if (tx < 15)
        gauss[tx] = computeExp(i2sigma2, tx);

    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;
    int px = xp & 15;
    int x = tx - px;

    // Load data from global memory
    for (int y = 0; y < 15; y++) {
        int xi = boundIndex(xp + x, w);
        int yi = boundIndex(yp + y, h);
        int memPos = 16 * y + x;
        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    // Compute gradients and update histograms
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = 16.0f * atan2f(dy, dx) / 3.1416f + 16.5f;
            if (bin == 32) bin = 0;
            float grad = sqrtf(dx * dx + dy * dy);
            hist[32 * (x - 1) + bin] += grad * gauss[x] * gauss[y];
        }
    }
    __syncthreads();

    // Optimize accumulation process using loop unrolling
    for (int y = 0; y < 5; y++)
        hist[y * 32 + tx] += hist[(y + 8) * 32 + tx];
    __syncthreads();

    for (int y = 0; y < 4; y++)
        hist[y * 32 + tx] += hist[(y + 4) * 32 + tx];
    __syncthreads();

    for (int y = 0; y < 2; y++)
        hist[y * 32 + tx] += hist[(y + 2) * 32 + tx];
    __syncthreads();

    hist[tx] += hist[32 + tx];
    __syncthreads();

    // Smooth histogram using a shared memory loop
    if (tx == 0)
        hist[32] = 6 * hist[0] + 4 * (hist[1] + hist[31]) + (hist[2] + hist[30]);
    if (tx == 1)
        hist[33] = 6 * hist[1] + 4 * (hist[2] + hist[0]) + (hist[3] + hist[31]);
    
    if (tx >= 2 && tx <= 29)
        hist[tx + 32] = 6 * hist[tx] + 4 * (hist[tx + 1] + hist[tx - 1]) +
                        (hist[tx + 2] + hist[tx - 2]);
    
    if (tx == 30)
        hist[62] = 6 * hist[30] + 4 * (hist[31] + hist[29]) + (hist[0] + hist[28]);
    if (tx == 31)
        hist[63] = 6 * hist[31] + 4 * (hist[0] + hist[30]) + (hist[1] + hist[29]);
    __syncthreads();

    // Find peaks in histogram
    float v = hist[32 + tx];
    hist[tx] = (v > hist[32 + ((tx + 1) & 31)] && v >= hist[32 + ((tx + 31) & 31)] ? v : 0.0f);
    __syncthreads();

    // Determine maximum values and orientations
    if (tx == 0) {
        float maxval1 = 0.0f;
        float maxval2 = 0.0f;
        int i1 = -1, i2 = -1;

        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }

        float val1 = hist[32 + ((i1 + 1) & 31)];
        float val2 = hist[32 + ((i1 + 31) & 31)];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Orient[bx] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);

        if (maxval2 < 0.8f * maxval1)
            i2 = -1;

        if (i2 >= 0) {
            val1 = hist[32 + ((i2 + 1) & 31)];
            val2 = hist[32 + ((i2 + 31) & 31)];
            float peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Orient[bx + maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        } else {
            d_Orient[bx + maxPts] = i2;
        }
    }
}