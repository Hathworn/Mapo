#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOrientations(float *g_Data, float *d_Sift, int maxPts, int w, int h)
{
  __shared__ float data[16*15];
  __shared__ float hist[32];
  __shared__ float gauss[16];
  
  const int tx = threadIdx.x;
  const int bx = blockIdx.x;
  
  // Initialize histogram and load Gaussian values
  if (tx < 32) hist[tx] = 0.0f;
  if (tx < 15) gauss[tx] = expf(-1.0f/(2.0f*3.0f*3.0f)*(tx-7)*(tx-7));
  __syncthreads();
  
  int xp = max(min(static_cast<int>(d_Sift[bx + 0*maxPts] - 6.5f), w - 1), 0);
  int yp = max(min(static_cast<int>(d_Sift[bx + 1*maxPts] - 6.5f), h - 1), 0);
  int px = xp & 15;
  int x = tx - px;
  
  // Load data into shared memory
  for (int y=0; y<15; y++) {
    int memPos = 16*y + x;
    int xi = max(min(xp + x, w - 1), 0);
    int yi = max(min(yp + y, h - 1), 0);
    if (x >= 0 && x < 15)
      data[memPos] = g_Data[yi*w + xi];
  }
  __syncthreads();
  
  // Compute gradient and populate histogram
  for (int y=1; y<14; y++) {
    int memPos = 16*y + x;
    if (x >= 1 && x < 14) {
      float dy = data[memPos+16] - data[memPos-16];
      float dx = data[memPos+1] - data[memPos-1];
      int bin = static_cast<int>(16.0f * atan2f(dy, dx) / 3.1416f + 16.5f) % 32;
      float grad = sqrtf(dx*dx + dy*dy);
      atomicAdd(&hist[bin], grad * gauss[x] * gauss[y]);
    }
  }
  __syncthreads();
  
  // Find the peaks in the histogram
  if (tx < 32) {
    float v = hist[tx];
    hist[tx] = (v > hist[(tx + 1) % 32] && v >= hist[(tx + 31) % 32]) ? v : 0.0f;
  }
  __syncthreads();
  
  // Determine dominant orientations
  if (tx == 0) {
    float maxval1 = 0.0f;
    float maxval2 = 0.0f;
    int i1 = -1;
    int i2 = -1;
    for (int i = 0; i < 32; i++) {
      float v = hist[i];
      if (v > maxval1) {
        maxval2 = maxval1;
        maxval1 = v;
        i2 = i1;
        i1 = i;
      } else if (v > maxval2) {
        maxval2 = v;
        i2 = i;
      }
    }
  
    float val1 = hist[(i1 + 1) % 32];
    float val2 = hist[(i1 + 31) % 32];
    float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
    d_Sift[bx + 5*maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
  
    if (maxval2 < 0.8f * maxval1)
      i2 = -1;
  
    if (i2 >= 0) {
      float val1 = hist[(i2 + 1) % 32];
      float val2 = hist[(i2 + 31) % 32];
      float peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
      d_Sift[bx + 6*maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
    } else {
      d_Sift[bx + 6*maxPts] = i2;
    }
  }
}