#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];

    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;

    if (x < width) {
        float val = d_Data[y * width + x];
        minvals[tx] = val;
        maxvals[tx] = val;

        // Loop unrolling to reduce branch instructions
        #pragma unroll
        for (int ty = 1; ty < 16; ty++) {
            val = d_Data[(y + ty) * width + x];
            if (val < minvals[tx]) minvals[tx] = val;
            if (val > maxvals[tx]) maxvals[tx] = val;
        }
    } else {
        // Handle x out of bounds with minimal impact
        minvals[tx] = FLT_MAX;
        maxvals[tx] = -FLT_MAX;
    }

    __syncthreads();

    // Parallel reduction for min and max
    for (int d = 64; d > 0; d >>= 1) {
        if (tx < d) {
            if (minvals[tx + d] < minvals[tx]) minvals[tx] = minvals[tx + d];
            if (maxvals[tx + d] > maxvals[tx]) maxvals[tx] = maxvals[tx + d];
        }
        __syncthreads();
    }

    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}