#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel(
    int32_t n, 
    float prior_offset, 
    float* cur_tot_weight, 
    int32_t max_count, 
    float* quadratic, 
    float* linear) 
{
    // Cache value to reduce repeated global memory reads
    float cur_weight = *cur_tot_weight;
    float val = 1.0f;

    if (max_count > 0) {
        float new_scale = max(cur_weight, static_cast<float>(max_count)) / max_count;
        val += (new_scale - 1.0f);
    }

    // Use a single loop iteration with a stride
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t offset = blockDim.x * gridDim.x;
    while (idx < n) {
        int32_t diag_idx = ((idx + 1) * (idx + 2) / 2) - 1;
        quadratic[diag_idx] += val;
        idx += offset;
    }

    // Use atomic addition for race-free update
    if (threadIdx.x == 0) {
        atomicAdd(&linear[0], val * prior_offset);
    }
}