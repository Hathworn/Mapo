#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* __restrict__ feats, int32_t ldf, float* __restrict__ feats_sq, int32_t lds) {
    int i = blockIdx.y * blockDim.y + threadIdx.y;  // Calculate global row index
    int j = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global column index

    // Ensure indices are within bounds
    if (i < num_rows && j < num_cols) {
        float f = feats[i * ldf + j];  // Load feature element
        feats_sq[i * lds + j] = f * f; // Store squared value
    }
}