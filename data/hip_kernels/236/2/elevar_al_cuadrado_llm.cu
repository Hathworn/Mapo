#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elevar_al_cuadrado(float * d_salida, float * d_entrada){
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Modify index calculation for full grid
    if (idx < N) { // Add boundary check to prevent out-of-bounds
        float f = d_entrada[idx];
        d_salida[idx] = f * f;
    }
}