#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hola()
{
    // Use threadIdx.x to display thread ID as well
    printf("Hola mundo! Soy un thread %d en el bloque %d\n", threadIdx.x, blockIdx.x);
}