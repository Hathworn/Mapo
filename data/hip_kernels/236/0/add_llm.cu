#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define N 100

__global__ void add(int *a, int *c)
{
    // Calculate unique thread index
    int tID = blockIdx.x * blockDim.x + threadIdx.x; 
    if (tID < N) // Ensure index is within bounds
    {
        c[tID] = 3 * a[tID];
    }
}