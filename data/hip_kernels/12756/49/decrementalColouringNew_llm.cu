#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void decrementalColouringNew(int *vertexArray, int *neighbourArray, int n, int m, int *decrementalArray, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Exit if the thread index exceeds the size
    if (i >= size) {
        return;
    }

    int startStart, startStop;
    int me, you;

    // Determine the vertices 'me' and its paired 'you'
    me = decrementalArray[i];
    you = (i % 2 == 0) ? decrementalArray[i + 1] : decrementalArray[i - 1];

    // Calculate the start and stop index for the vertex's neighbors
    startStart = vertexArray[me - 1];
    startStop = vertexArray[me];

    // Use a single loop to find and zero out the neighbor
    for (int j = startStart; j < startStop; j++) {
        if (neighbourArray[j] == you) {
            neighbourArray[j] = 0;
            break; // Exit once the match is found and processed
        }
    }
}