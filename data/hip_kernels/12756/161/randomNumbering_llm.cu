#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomNumbering(hiprandState* globalState, int *degreeCount, int n, int limit) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) { // Ensure we don't access out of bounds
        hiprandState localState = globalState[i];
        float RANDOM = hiprand_uniform(&localState);
        RANDOM = RANDOM * limit + 1; // Removed unnecessary operations
        degreeCount[i] = (int)RANDOM;
        globalState[i] = localState; // Update state after computation
    }
}