#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixColour(float *a, float *b, int n) {
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    if (j < n) {
        for (int i = 0; i < n; i++) {
            // Minimize printf usage for better performance
            if (a[j * n + i] == 1) {
                if (b[j] == b[i]) {
                    b[j] = -1;
                    break;
                }
            }
        }
    }
}