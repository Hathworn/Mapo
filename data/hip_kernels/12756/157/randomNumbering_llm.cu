#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomNumbering(hiprandState* globalState, int *degreeCount, int n, int limit) {
    // Use shared memory for block-level state and pre-calculate expressions
    extern __shared__ hiprandState sharedState[];
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        if (threadIdx.x == 0) {
            sharedState[blockDim.x] = globalState[blockIdx.x];
        }
        __syncthreads();

        hiprandState localState = sharedState[threadIdx.x];
        float RANDOM = hiprand_uniform(&localState);

        sharedState[threadIdx.x] = localState;
        globalState[i] = sharedState[threadIdx.x];

        float scale_factor = limit - 0.000001f;
        RANDOM = RANDOM * scale_factor + 1.0f; // Pre-calculate scale factor

        degreeCount[i] = (int)RANDOM;
    }
}