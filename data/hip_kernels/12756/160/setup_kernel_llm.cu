#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Using the hipLaunchBounds macro to optimize occupancy
__global__ void __launch_bounds__(256, 4) setup_kernel(hiprandState *state, unsigned long seed) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimized to use fewer registers and increase efficiency
    if (i < gridDim.x * blockDim.x) {
        hiprand_init(seed, i, 0, &state[i]);
    }
}