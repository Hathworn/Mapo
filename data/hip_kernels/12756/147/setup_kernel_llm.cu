#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState * state, unsigned long seed)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Initialize with a persistent random state per thread
    hiprandState localState;
    hiprand_init(seed, i, 0, &localState);
    
    // Store the local state into global memory
    state[i] = localState;
}