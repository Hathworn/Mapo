#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_kernel (hiprandState * state, unsigned long seed )
{
int i= blockDim.x * blockIdx.x + threadIdx.x;

hiprand_init (seed, i, 0, &state[i]);
}