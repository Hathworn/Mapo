#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void vectorAddition (float *a, float *b, float *c, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Check within bounds and perform addition
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}