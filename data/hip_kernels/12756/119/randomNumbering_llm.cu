#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomNumbering (hiprandState* globalState, int *degreeCount, int n, int limit) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n) return; // Boundary check for thread index

    hiprandState localState = globalState[i];
    float RANDOM = hiprand_uniform(&localState) * (limit - 1) + 1; // Combine operations

    degreeCount[i] = (int)RANDOM;
    globalState[i] = localState; // Move state update after calculation
}