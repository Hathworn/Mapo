#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimized for better memory and performance
__global__ void randomColouring(hiprandState* globalState, int* degreeCount, int n, int limit) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < n) { // Ensure valid thread index
        // Use localState to avoid repeated global memory accesses
        hiprandState localState = globalState[i];
        float RANDOM = hiprand_uniform(&localState);

        RANDOM = RANDOM * (limit - 0.000001f) + 1.0f; // Adjust random range calculation

        degreeCount[i] = static_cast<int>(RANDOM);

        globalState[i] = localState; // Update global state with modified local state
    }
}