#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomNumbering(hiprandState* globalState, int *degreeCount, int n, int limit) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (i >= n) return;

    hiprandState localState = globalState[i];
    float RANDOM = hiprand_uniform(&localState);
    globalState[i] = localState;

    // Optimize random number calculation by avoiding unnecessary operations
    RANDOM = RANDOM * (limit - 1) + 1;

    degreeCount[i] = (int)RANDOM;
}