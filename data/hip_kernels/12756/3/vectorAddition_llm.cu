#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAddition(float *a, float *b, float *c, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Use shared memory for faster data access
    __shared__ float a_shared[256];
    __shared__ float b_shared[256];

    if (i < n) {
        // Ensure i is valid for shared memory index
        int local_i = threadIdx.x;
        
        // Load data into shared memory
        a_shared[local_i] = a[i];
        b_shared[local_i] = b[i];
        __syncthreads(); // Synchronize threads

        // Compute with shared memory
        c[i] = a_shared[local_i] + b_shared[local_i];
    }
}