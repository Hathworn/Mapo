#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conflictDetection(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m, int *detectConflict) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure thread index is within bounds
    if (i >= n) {
        return;
    }

    int myColour = degreeCount[i];
    int start = vertexArray[i];
    int stop = (i == n - 1) ? 2 * m : vertexArray[i + 1];

    // Use shared memory for conflict detection
    extern __shared__ int sharedDetectConflict[];
    if (threadIdx.x == 0) {
        sharedDetectConflict[0] = 0;
    }
    __syncthreads();

    // Iterate over all neighbors
    for (int j = start; j < stop; j++) {
        if (degreeCount[neighbourArray[j] - 1] == myColour) { // Color conflict detected
            int neighborIndex = neighbourArray[j] - 1;
            
            atomicOr(&detectConflict[i], (i < neighborIndex)); // Set conflict for current vertex
            atomicOr(&detectConflict[neighborIndex], (i >= neighborIndex)); // Set conflict for neighbor
        }
    }

    // Final synchronization for shared memory
    __syncthreads();
}