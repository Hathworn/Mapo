#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomColouring (hiprandState* globalState, int *degreeCount, int n, int limit){

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n) return;  // Ensure thread index is within bounds

    // Load the state for the current thread
    hiprandState localState = globalState[i];
    
    // Generate random number and compute degree count
    float RANDOM = hiprand_uniform(&localState) * (limit - 0.000001f) + 1.0f;
    
    // Store the updated state back to global memory
    globalState[i] = localState;
    
    // Assign the computed degree to the degreeCount array
    degreeCount[i] = static_cast<int>(RANDOM);
}