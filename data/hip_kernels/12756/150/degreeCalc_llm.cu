#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int sum = 1;

__global__ void degreeCalc (int *array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= 1000000) {
        return;
    }

    // Atomic addition to prevent race conditions
    atomicAdd(&sum, array[i]);

    // Uncomment if printing is required
    // if (i == 999999) {
    //     printf("%d", sum);
    // }
}

__global__ void degreeCalc (int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= n) {
        return;
    }

    // Directly calculate and assign the degree
    degreeCount[i] = vertexArray[i + 1] - vertexArray[i];
}