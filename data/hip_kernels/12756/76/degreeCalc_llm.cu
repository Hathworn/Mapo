#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_THREADS_PER_BLOCK 1024
__device__ int sum = 1;

__global__ void degreeCalc(int *array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Bounds check optimized to prevent unnecessary calculations
    if (i >= 1000000) return;

    // Atomic operation to prevent race conditions
    atomicAdd(&sum, array[i]);

    // Printing is removed for performance optimization
}

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Bounds check optimized to prevent unnecessary calculations
    if (i >= n) return;

    // Simplified calculation with fewer variables
    degreeCount[i] = vertexArray[i + 1] - vertexArray[i];
}