#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    // Initialize random state using a unique seed for each thread
    hiprand_init(seed, id, 0, &state[id]);
}