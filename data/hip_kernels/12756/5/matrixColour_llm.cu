#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixColour (float *a, float *b, int n) {
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    // Remove unnecessary prints to reduce overhead
    // printf("Block = %d ; Thread = %d \n", blockIdx.x+1, threadIdx.x+1);

    int *colour = new int[n];
    memset(colour, 0, n * sizeof(int));

    if (j < n) {
        for (int i = 0; i < n; i++) {
            // printf("Block = %d ; Thread = %d First For i = %d\n", blockIdx.x+1, threadIdx.x+1, i+1);
            if (a[j * n + i] == 1 && b[i] != -1) {
                colour[(int)b[i]] = 1;
            }
        }

        for (int i = 0; i < n; i++) {
            if (colour[i] == 0) {
                // atomic operations should be minimized; use directly if possible
                b[j] = (float)i;
                // printf("Block = %d ; Thread = %d Second For i = %d\n", blockIdx.x+1, threadIdx.x+1, i+1);
                break;
            }
        }
    }

    delete[] colour;
}