#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sum = 1;  

__global__ void degreeCalc(int *array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure that only valid indices are processed with improved condition check
    if (i < 1000000) {
        atomicAdd(&sum, array[i]); // Use atomic operation to safely update 'sum'
    }
}

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Execute thread only when 'i' is within range
    if (i < n) {
        int start = vertexArray[i];
        int stop = (i == n - 1) ? 2 * m : vertexArray[i + 1]; // Improved conditional assignment
        degreeCount[i] = stop - start; // Directly calculate degree count
    }
}