#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel (hiprandState * state, unsigned long seed) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimize by removing unnecessary variables and improving readability
    if (idx < gridDim.x * blockDim.x) {
        hiprand_init(seed, idx, 0, &state[idx]);
    }
}