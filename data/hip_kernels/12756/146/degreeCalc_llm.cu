#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sum = 1;

__global__ void degreeCalc(int *array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < 1000000) {
        atomicAdd(&sum, array[i]); // Use atomic operation for safety in parallel environment
    }
}

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        int start = vertexArray[i];
        int stop = (i == n - 1) ? 2 * m : vertexArray[i + 1]; // Simplify stop calculation
        degreeCount[i] = stop - start;
    }
}