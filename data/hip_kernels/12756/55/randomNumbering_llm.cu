#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomNumbering (hiprandState* globalState, int *degreeCount, int n, int limit) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) { // Bounds check
        hiprandState localState = globalState[i];
        float RANDOM = hiprand_uniform(&localState);
        globalState[i] = localState;

        // Reduce number of operations in transformation
        RANDOM = RANDOM * limit + (1.0f - RANDOM);

        degreeCount[i] = (int) RANDOM;
    }
}