#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void degreeCalc(int *array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= 1000000) {
        return;
    }

    // Use shared memory for reduction
    __shared__ int localSum[256];  // Example: 256 threads per block, adjust if needed
    localSum[threadIdx.x] = array[i];
    __syncthreads();

    // Parallel reduction within the block
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            localSum[threadIdx.x] += localSum[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Add the block's result to global sum
    if (threadIdx.x == 0) {
        atomicAdd(&sum, localSum[0]);
    }
}

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= n) {
        return;
    }

    int start = vertexArray[i];
    int stop = (i == n - 1) ? 2 * m : vertexArray[i + 1];
    degreeCount[i] = stop - start;
}