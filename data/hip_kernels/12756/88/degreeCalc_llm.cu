#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sum = 1;

__global__ void degreeCalc(int *array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Return early for out-of-bounds indices
    if (i >= 1000000) {
        return;
    }
    // Use atomic addition to prevent race conditions
    atomicAdd(&sum, array[i]);
}

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Return early for out-of-bounds indices
    if (i >= n) {
        return;
    }
    // Calculate degree using start and stop indices
    int start = vertexArray[i];
    int stop = vertexArray[i + 1];
    degreeCount[i] = stop - start;
}