#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize by ensuring state is initialized only for valid indices
__global__ void setup_kernel (hiprandState * state, unsigned long seed ){

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < gridDim.x * blockDim.x) { // Ensure index is within valid range
        hiprand_init(seed, i, 0, &state[i]);
    }
}