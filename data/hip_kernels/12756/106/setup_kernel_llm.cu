#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState* __restrict__ state, unsigned long seed) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Directly initializing RNG state without additional offset
    hiprand_init(seed, i, 0, state + i);
}