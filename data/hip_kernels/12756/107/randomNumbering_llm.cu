#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomNumbering (hiprandState* globalState, int *degreeCount, int n, int limit) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Only proceed if i is within bounds to avoid out-of-bounds memory access
    if (i < n) {
        hiprandState localState = globalState[i];
        float RANDOM = hiprand_uniform(&localState);
        globalState[i] = localState; // Update the global state with the local state

        RANDOM *= (limit - 1) + 0.999999f;
        RANDOM += 1.0f;

        degreeCount[i] = static_cast<int>(RANDOM);
    }
}