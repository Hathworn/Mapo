#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREAD_PER_BLOCK 256

// Optimized global kernel function
__global__ void degreeCalc(int *vertexArray, int *degreeCount, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        // Calculate degree count directly
        degreeCount[i] = vertexArray[i + 1] - vertexArray[i];
    }
}

// Original global kernel for sum computation
__device__ int sum = 1;
__global__ void computeSum(int *array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < 1000000) {
        atomicAdd(&sum, array[i]); // Use atomic to prevent race conditions
    }
}