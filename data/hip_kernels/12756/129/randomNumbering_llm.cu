#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved global kernel function
__global__ void randomNumbering(hiprandState* globalState, int *degreeCount, int n, int limit) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {  // Ensure thread index is within bounds
        hiprandState localState = globalState[i];
        float randomValue = hiprand_uniform(&localState);
        globalState[i] = localState;

        randomValue *= (limit - 1 + 0.999999);
        randomValue += 1;

        degreeCount[i] = static_cast<int>(randomValue); // Use static_cast for type conversion
    }
}