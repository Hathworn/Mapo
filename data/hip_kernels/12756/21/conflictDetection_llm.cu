#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conflictDetection(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m, int *detectConflict) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Exit if index i is out of vertex range
    if (i >= n) return;
    
    int myColour = degreeCount[i];
    int start = vertexArray[i];
    int stop = (i == n - 1) ? m : vertexArray[i + 1];
    
    // Use shared memory to reduce global memory access latency
    __shared__ int sharedDegree[1024]; // Adjust size based on max thread number per block
    sharedDegree[threadIdx.x] = myColour;
    __syncthreads();

    for (int j = start; j < stop; j++) {
        int neighbourIndex = neighbourArray[j] - 1;
        // Compare using shared memory to reduce global memory fetches
        if (sharedDegree[threadIdx.x] == degreeCount[neighbourIndex]) {
            detectConflict[i] = 1;
            break;
        }
    }
}