#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_kernel (hiprandState * state, unsigned long seed ) {
    // Calculate global index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure the thread index is within the valid range
    if (i < gridDim.x * blockDim.x) {
        // Initialize the random state
        hiprand_init(seed, i, 0, &state[i]);
    }
}