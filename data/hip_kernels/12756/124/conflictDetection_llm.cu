#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conflictDetection (int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m, int *detectConflict) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n) {
        return;
    }

    int myColour = degreeCount[i];
    int start = vertexArray[i];
    int stop = vertexArray[i + 1];

    for (int j = start; j < stop; j++) {
        int neighborIndex = neighbourArray[j] - 1;
        if (degreeCount[neighborIndex] == myColour) {
            if (i < neighborIndex) {
                // Atomic set to prevent race conditions
                atomicExch(&detectConflict[i], 1);
            } else {
                atomicExch(&detectConflict[neighborIndex], 1);
            }
        }
    }
}