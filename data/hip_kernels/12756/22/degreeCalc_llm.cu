#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sum = 1;

// Optimized global kernel function
__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if thread index is within bounds
    if (i >= n) return;

    // Load start and stop indices to shared memory to reduce global memory access
    __shared__ int start, stop;
    
    start = vertexArray[i];
    stop = (i == n - 1) ? m : vertexArray[i + 1];

    int diff = stop - start;
    
    // Atomic addition for degree count
    atomicAdd(&degreeCount[i], diff);

    // Loop through neighbors
    for (int j = start; j < stop; j++) {
        atomicAdd(&degreeCount[neighbourArray[j] - 1], 1);
    }
}