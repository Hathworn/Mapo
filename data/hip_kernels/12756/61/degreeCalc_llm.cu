#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 256

__global__ void degreeCalcOptimized(int *array, int array_size, int *result) {
    __shared__ int local_sum[BLOCK_SIZE];  // Shared memory for partial sums
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    local_sum[threadIdx.x] = 0;

    if (i < array_size) {
        local_sum[threadIdx.x] = array[i];
    }
    __syncthreads();

    // Reduce within block
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            local_sum[threadIdx.x] += local_sum[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Accumulate block results to global memory
    if (threadIdx.x == 0) {
        atomicAdd(result, local_sum[0]);
    }
}

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i >= n) {
        return;
    }
    
    int start = vertexArray[i];
    int stop = (i == n - 1) ? 2 * m : vertexArray[i + 1];
    degreeCount[i] = stop - start;
}