#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void degreeCalcOptimized(int *vertexArray, int *degreeCount, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        // Efficiently calculate degree using pre-fetched vertex points
        int start = vertexArray[i];
        int stop = vertexArray[i + 1];
        degreeCount[i] = stop - start;
    }
}