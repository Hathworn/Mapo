#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sum = 1;

__global__ void degreeCalc(int *array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < 1000000) {
        atomicAdd(&sum, array[i]);  // Use atomic operation to prevent data race
    }
}

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        // Use registers to store frequently used values
        int start = vertexArray[i];
        int stop = (i == n - 1) ? 2 * m : vertexArray[i + 1];
        int diff = stop - start;
        degreeCount[i] = diff;
    }
}