#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sum = 1;

__global__ void degreeCalc(int *array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Use a shared variable for partial sums
    __shared__ int local_sum;

    if (threadIdx.x == 0) local_sum = 0;
    __syncthreads();

    if (i < 1000000) {
        atomicAdd(&local_sum, array[i]); // Atomic addition to avoid race condition
    }

    __syncthreads();

    if (threadIdx.x == 0) atomicAdd(&sum, local_sum); // Combine partial sums
}

// Optimize by reducing redundant operations and using grid-stride loop
__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x) {
        int start = vertexArray[idx];
        int stop = (idx == n - 1) ? 2 * m : vertexArray[idx + 1];
        degreeCount[idx] = stop - start;
    }
}