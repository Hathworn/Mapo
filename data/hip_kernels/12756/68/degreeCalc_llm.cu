#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sum = 1;

__global__ void degreeCalc (int *array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory for better memory access
    __shared__ int localSum;
    if (threadIdx.x == 0) localSum = 0;
    __syncthreads();

    if (i < 1000000) {
        atomicAdd(&localSum, array[i]);
    }
    __syncthreads();

    // Reduce local sums and update global sum
    if (threadIdx.x == 0) atomicAdd(&sum, localSum);
}

__global__ void degreeCalc (int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        // Directly calculate degree count without redundant variables
        degreeCount[i] = vertexArray[i + 1] - vertexArray[i];
    }
}