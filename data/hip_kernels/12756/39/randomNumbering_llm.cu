#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomNumbering (hiprandState* globalState, int *degreeCount, int n, int limit) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure thread index is within bounds
    if (i < n) {
        hiprandState localState = globalState[i];
        float RANDOM = hiprand_uniform(&localState);
        globalState[i] = localState;

        // Perform calculation directly without repeated operations
        degreeCount[i] = (int)(RANDOM * limit + 1);
    }
}