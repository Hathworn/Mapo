#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void conflictDetection (int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m, int *detectConflict) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= n) {
        return;
    }

    int myColour = degreeCount[i];
    int start = vertexArray[i];
    int stop = (i == n - 1) ? 2 * m : vertexArray[i + 1];

    for (int j = start; j < stop; j++) {
        int neighborIdx = neighbourArray[j] - 1;
        if (degreeCount[neighborIdx] == myColour) {
            // Use atomic operation to ensure thread safety
            if (i < neighborIdx) {
                atomicExch(&detectConflict[i], 1);
            } else {
                atomicExch(&detectConflict[neighborIdx], 1);
            }
        }
    }
}