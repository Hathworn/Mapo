#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomNumbering (hiprandState* globalState, int *degreeCount, int n, int limit){

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if i is within bounds
    if (i < n) {
        hiprandState localState = globalState[i];
        float RANDOM = hiprand_uniform(&localState);
        globalState[i] = localState;

        RANDOM *= limit; // More efficient scaling operation
        RANDOM += 1.0f;

        degreeCount[i] = (int) RANDOM;
    }
}