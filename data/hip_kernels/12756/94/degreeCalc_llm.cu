#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sum = 1;

__global__ void degreeCalc (int *array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < 1000000) {
        atomicAdd(&sum, array[i]); // Use atomic operation for thread safety
        // Uncomment for debugging
        // if (i == 999999) {
        //     printf("%d", sum);
        // }
    }
}

__global__ void degreeCalc (int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        int start = vertexArray[i];
        int stop = vertexArray[i + 1];
        int diff = stop - start;

        degreeCount[i] = diff; // Directly compute degree difference
    }
}