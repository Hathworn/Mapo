#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x; // Use meaningful variable name 'idx'
    hiprand_init(seed, idx, 0, &state[idx]); // Align spacing for clarity
}