#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    // Use a more descriptive variable name and cache block and grid info
    int globalIndex = blockDim.x * blockIdx.x + threadIdx.x;
    hiprand_init(seed, globalIndex, 0, &state[globalIndex]);
}