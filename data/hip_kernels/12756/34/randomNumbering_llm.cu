#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel function
__global__ void randomNumbering(hiprandState* globalState, int* degreeCount, int n, int limit) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure valid index range
    if (i < n) {
        // Use local state for random number generation
        hiprandState localState = globalState[i];
        float RANDOM = hiprand_uniform(&localState);
        
        // Store updated state back to global memory
        globalState[i] = localState;
        
        // Calculate random number within specific range
        RANDOM = RANDOM * (limit - 1) + 1;
        
        // Store result in degreeCount array
        degreeCount[i] = static_cast<int>(RANDOM);
    }
}