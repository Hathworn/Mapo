#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize: Ensure state accesses are coalesced and avoid unnecessary index computation
__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, i, 0, &state[i]);
}