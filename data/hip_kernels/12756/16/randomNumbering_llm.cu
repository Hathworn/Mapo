#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomNumbering (hiprandState* globalState, int *degreeCount, int n, int limit){

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {  // Ensure within bounds (grid-stride loop can be added for more efficiency)
        hiprandState localState = globalState[i];
        float RANDOM = hiprand_uniform(&localState);
        
        // Scale and shift the random number
        RANDOM = RANDOM * limit + 1.0f;

        degreeCount[i] = (int) RANDOM;
        
        globalState[i] = localState; // Ensure updated state is stored
    }
}