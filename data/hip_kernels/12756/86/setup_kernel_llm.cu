#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    // Calculate the global index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimize for warp efficiency: initialize state for each thread within the warp
    if (i < gridDim.x * blockDim.x) {
        // Initialize the random state
        hiprand_init(seed, i, 0, &state[i]);
    }
}