#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomColouring(hiprandState* globalState, int* degreeCount, int n, int limit) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < n) {  // Ensure index i is within bounds
        // Use shared memory to avoid redundant global memory access
        __shared__ hiprandState localState;
        localState = globalState[i];
        float RANDOM = hiprand_uniform(&localState);
        
        RANDOM = RANDOM * limit + 1; // Calculate directly in one line
        degreeCount[i] = (int)RANDOM;
        
        globalState[i] = localState; // Save state back to global
    }
}