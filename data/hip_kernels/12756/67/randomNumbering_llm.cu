#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomNumbering(hiprandState* globalState, int *degreeCount, int n, int limit) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) { // Ensure access is within bounds
        hiprandState localState = globalState[i];
        float RANDOM = hiprand_uniform(&localState);
        globalState[i] = localState;

        RANDOM *= (limit - 1 + 0.999999);
        RANDOM += 1;

        degreeCount[i] = static_cast<int>(RANDOM); // Use C++ casting
    }
}