#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void graphGenerate (float *a, float *b, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure grid size doesn't exceed data size.
    if (i < n) {
        // Avoid recomputation inside the loop.
        int threadIndex = threadIdx.x;
        a[i] = threadIndex * 2;
        b[i] = threadIndex;
    }
}