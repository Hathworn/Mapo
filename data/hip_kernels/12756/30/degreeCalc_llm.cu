#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sum = 1;  

__global__ void degreeCalc(int *array) {
    // Calculate global index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Check boundary condition
    if (i >= 1000000) {
        return;
    }

    // Use atomic operation to prevent race condition
    atomicAdd(&sum, array[i]);
}

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    // Calculate global index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Check boundary condition
    if (i >= n) {
        return;
    }

    // Calculate start and stop indices, compute degree
    int start = vertexArray[i];
    int stop = (i == n - 1) ? 2 * m : vertexArray[i + 1];
    int diff = stop - start;

    // Save computed degree
    degreeCount[i] = diff;
}