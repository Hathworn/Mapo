#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomNumbering(hiprandState* globalState, int *degreeCount, int n, int limit) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure thread index doesn't exceed number of elements
    if (i < n) {
        hiprandState localState = globalState[i];
        float RANDOM = hiprand_uniform(&localState);
        globalState[i] = localState;

        // Precompute constant factor outside loop
        float factor = limit - 1 + 0.999999;
        
        RANDOM *= factor;
        RANDOM += 1;

        degreeCount[i] = (int) RANDOM;
    }
}