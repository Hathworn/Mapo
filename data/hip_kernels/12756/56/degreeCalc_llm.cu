#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sum = 1;

__global__ void degreeCalc (int *array) {
    // Calculate the global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Bounds check to ensure index falls within valid range
    if (i >= 1000000) {
        return;
    }

    // Use atomic operation to prevent race conditions
    atomicAdd(&sum, array[i]);
}

__global__ void degreeCalc (int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    // Calculate the global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Bounds check to ensure index falls within valid range
    if (i >= n) {
        return;
    }

    // Efficiently compute the degree by directly calculating the difference
    int start = vertexArray[i];
    int stop = vertexArray[i + 1];
    degreeCount[i] = stop - start;
}