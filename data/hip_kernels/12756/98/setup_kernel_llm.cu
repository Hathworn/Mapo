#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Check to ensure within state array bounds
    if (i < gridDim.x * blockDim.x) {
        hiprand_init(seed, i, 0, &state[i]);
    }
}