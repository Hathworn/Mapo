#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int sum = 1;

__global__ void degreeCalc(int* array) {
    // Calculate thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Efficient boundary check
    if (i >= 1000000) return;

    // Atomic addition for concurrent updates
    atomicAdd(&sum, array[i]);
}

__global__ void degreeCalc(int* vertexArray, int* neighbourArray, int* degreeCount, int n, int m) {
    // Calculate thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Efficient boundary check
    if (i >= n) return;

    // Calculate start and stop indices
    int start = vertexArray[i];
    int stop = (i == n-1) ? 2 * m : vertexArray[i + 1];

    // Store the degree count
    degreeCount[i] = stop - start;
}