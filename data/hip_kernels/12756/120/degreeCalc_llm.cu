#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void degreeCalc(int *array) {
    // Initialize shared memory for thread-safe accumulation
    extern __shared__ int sharedSum[];
    int tid = threadIdx.x;
    int i = blockDim.x * blockIdx.x + tid;

    sharedSum[tid] = (i < 1000000) ? array[i] : 0;
    __syncthreads();

    // Reduce within block
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sharedSum[tid] += sharedSum[tid + stride];
        }
        __syncthreads();
    }

    // Only one thread updates the global sum
    if (tid == 0) {
        atomicAdd(&sum, sharedSum[0]);
    }
}

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure i is within bounds
    if (i < n) {
        int start = vertexArray[i];
        int stop = vertexArray[i + 1];
        degreeCount[i] = stop - start; // Directly calculate diff within the assignment
    }
}