#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized setup_kernel function
__global__ void setup_kernel (hiprandState * state, unsigned long seed )
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Non-conflicting register name
    hiprand_init(seed, idx, 0, &state[idx]); // Direct indexing without unnecessary variable
}