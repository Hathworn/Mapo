#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomNumbering (hiprandState* globalState, int *degreeCount, int n, int limit) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < n) { // Ensure index is within bounds
        hiprandState localState = globalState[i];
        float RANDOM = hiprand_uniform(&localState);
        globalState[i] = localState;  // Save local state back to global
        
        // Optimize random number scaling
        RANDOM = RANDOM * limit + 1;
        
        degreeCount[i] = (int) RANDOM;
    }
}