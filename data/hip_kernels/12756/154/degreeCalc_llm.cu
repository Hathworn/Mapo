#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sum = 1;

// Improved: Utilize atomic operation for safe concurrent updates
__global__ void degreeCalc(int *array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= 1000000) {
        return;
    }

    atomicAdd(&sum, array[i]);  // Atomic addition to avoid race condition
}

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= n) {
        return;
    }

    int start = vertexArray[i];
    int stop = vertexArray[i + 1];
    int diff = stop - start;

    degreeCount[i] = diff;  // Assign difference directly
}