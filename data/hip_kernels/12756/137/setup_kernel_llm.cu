#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    // Use the fastest way to calculate the global index
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    // Initialize the random state for each thread
    hiprand_init(seed, i, 0, &state[i]);
}