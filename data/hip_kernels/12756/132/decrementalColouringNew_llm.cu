#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void decrementalColouringNew(int *vertexArray, int *neighbourArray, int n, int m, int *decrementalArray, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Early termination for threads exceeding bounds
    if (i >= size) return;

    int startStart, startStop;
    int me, you;

    me = decrementalArray[i];
    you = (i % 2 == 0) ? decrementalArray[i + 1] : decrementalArray[i - 1];

    printf("I am %d and I am deleting %d - %d\n", i, me, you);

    // Determine start and stop based on vertex
    startStart = vertexArray[me - 1];
    startStop = (me == n) ? 2 * m : vertexArray[me];

    // Optimize loop to exit once you is found
    for (int j = startStart; j < startStop; ++j) {
        if (neighbourArray[j] == you) {
            neighbourArray[j] = 0;
            break;
        }
    }
}