#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorValue (float *a, float *b, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds prior to performing operations
    if (i < n) {
        // Combine calculations to reduce redundant accesses
        float val = static_cast<float>(threadIdx.x);
        a[i] = val * 2.0f;
        b[i] = val;
    }
}