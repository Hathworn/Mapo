#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sum = 1;

__global__ void degreeCalc(int *array) {
    // Use shared memory to reduce global memory writes
    __shared__ int localSum;
    if (threadIdx.x == 0) {
        localSum = 0;
    }
    __syncthreads();

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < 1000000) {
        atomicAdd(&localSum, array[i]); // Use atomicAdd to avoid race conditions
    }
    __syncthreads();

    // Only the first thread adds to global sum to avoid contention
    if (threadIdx.x == 0) {
        atomicAdd(&sum, localSum);
    }
}

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        int start = vertexArray[i];
        int stop = (i == n - 1) ? (2 * m) : vertexArray[i + 1];
        
        // Store degree difference directly to avoid additional variables
        degreeCount[i] = stop - start;
    }
}