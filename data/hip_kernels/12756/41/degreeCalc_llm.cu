#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sum = 1;

// Optimized version: parallel reduction to avoid contention
__global__ void degreeCalcOptimized(int *array, int length) {
    extern __shared__ int sharedSum[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int localSum = 0;

    if (i < length) {
        localSum = array[i];
    }

    sharedSum[threadIdx.x] = localSum;
    __syncthreads();

    // Perform parallel reduction
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sharedSum[threadIdx.x] += sharedSum[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        atomicAdd(&sum, sharedSum[0]);
    }
}

// Improved atomic operations to minimize contention
__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        int start = vertexArray[i];
        int stop = (i == n - 1) ? m : vertexArray[i + 1];
        int diff = stop - start;

        atomicAdd(&degreeCount[i], diff);

        for (int j = start; j < stop; j++) {
            atomicAdd(&degreeCount[neighbourArray[j] - 1], 1);
        }
    }
}