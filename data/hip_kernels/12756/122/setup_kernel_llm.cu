#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState *state, unsigned long seed)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a single call to obtain the block index
    if (id < gridDim.x * blockDim.x) {
        hiprand_init(seed, id, 0, &state[id]);
    }
}