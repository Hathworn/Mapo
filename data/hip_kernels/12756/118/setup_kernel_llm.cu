#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimize by using faster atomic operations for initializing states
    if (i < gridDim.x * blockDim.x) {
        hiprand_init(seed, i, 0, &state[i]);
    }
}