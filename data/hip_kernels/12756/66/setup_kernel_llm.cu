#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized setup_kernel function
__global__ void setup_kernel (hiprandState *state, unsigned long seed) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < gridDim.x * blockDim.x) { // Ensure within bounds
        hiprand_init(seed, i, 0, &state[i]);
    }
}