#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    // Calculate global thread index using block and thread indices
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Initialize RNG state with a unique seed per thread
    hiprand_init(seed, i, 0, &state[i]);
}