#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global thread index

    // Initialize the random state with given seed
    if (i < gridDim.x * blockDim.x) {
        hiprand_init(seed, i, 0, &state[i]); // Ensure state index is within bounds
    }
}