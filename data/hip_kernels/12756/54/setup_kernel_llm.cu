#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function
__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure the index is within bounds before initializing
    if (i < gridDim.x * blockDim.x) {
        hiprand_init(seed, i, 0, &state[i]);
    }
}