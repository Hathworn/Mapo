#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void decrementalColouringNew(int *vertexArray, int *neighbourArray, int n, int m, int *decrementalArray, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Early exit for out of bounds threads
    if (i >= size) {
        return;
    }
    
    int me = decrementalArray[i];
    int you = (i % 2 == 0) ? decrementalArray[i + 1] : decrementalArray[i - 1];
    
    // Calculate start and stop indices
    int startStart = vertexArray[me - 1];
    int startStop = vertexArray[me];
    
    // Replace nested loop with efficient search
    for (int j = startStart; j < startStop; j++) {
        if (neighbourArray[j] == you) {
            neighbourArray[j] = 0; // Mark as deleted
            break;
        }
    }
}