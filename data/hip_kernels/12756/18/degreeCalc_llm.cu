#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sum = 1;

// Optimized kernel with unrolls for compute efficiency
__global__ void degreeCalc(int *array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < 1000000) {
        atomicAdd(&sum, array[i]); // Use atomic add for safe global updates
    }
}

// Optimized kernel with reduced branching
__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        int start = vertexArray[i];
        int stop = (i == n - 1) ? m : vertexArray[i + 1]; // Conditional operator for branch reduction

        int diff = stop - start;

        atomicAdd(&degreeCount[i], diff); // Atomic for concurrency control

        for (int j = start; j < stop; j++) {
            atomicAdd(&degreeCount[neighbourArray[j] - 1], 1); // Atomic for concurrent additions
        }
    }
}