#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel (hiprandState * state, unsigned long seed){

    // Calculate the unique thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    /* Optimize kernel by eliminating branch divergence
       Check if thread index is within state size */
    if (i < gridDim.x * blockDim.x) {
        // Initialize the random state for each thread
        hiprand_init(seed, i, 0, &state[i]);
    }
}