#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomNumbering(hiprandState* globalState, int* degreeCount, int n, int limit) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure we don't access memory out of bounds
    if (i < n) {  
        hiprandState localState = globalState[i];
        
        // Directly use the calculated random number
        float RANDOM = hiprand_uniform(&localState) * (limit - 1 + 0.999999) + 1;
        
        globalState[i] = localState;
        
        degreeCount[i] = (int)RANDOM;
    }
}