#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void degreeCalc(int *array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= 1000000) {
        return;
    }
    atomicAdd(&sum, array[i]); // Optimize sum operation with atomicAdd
}

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= n) {
        return;
    }
    
    int start = vertexArray[i];
    int stop = (i == n - 1) ? 2 * m : vertexArray[i + 1]; // Simplified conditional expression
    degreeCount[i] = stop - start;
}