#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomColouring (hiprandState* globalState, int *degreeCount, int n, int limit) {
    // Calculate the global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure we don't access out-of-bounds memory
    if (i < n) {
        // Read the state, generate a random number, and write back the state
        hiprandState localState = globalState[i];
        float RANDOM = hiprand_uniform(&localState) * (limit - 1 + 0.999999) + 1;
        globalState[i] = localState;

        // Convert float to int and store in degreeCount
        degreeCount[i] = (int)RANDOM;
    }
}
```
