#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomNumbering (hiprandState* globalState, int *degreeCount, int n, int limit) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) { // Ensure i is within bounds
        hiprandState localState = globalState[i];
        float RANDOM = hiprand_uniform(&localState);
        globalState[i] = localState;

        RANDOM = RANDOM * (limit - 0.000001f) + 1; // Simplify calculation
        degreeCount[i] = (int)RANDOM;
    }
}