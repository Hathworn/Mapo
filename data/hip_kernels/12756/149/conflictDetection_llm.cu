#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conflictDetection(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m, int *detectConflict) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= n) {
        return;
    }

    int myColour = degreeCount[i];
    int start = vertexArray[i];
    int stop = vertexArray[i + 1];

    for (int j = start; j < stop; j++) {
        int neighborIdx = neighbourArray[j] - 1;
        
        if (degreeCount[neighborIdx] == myColour) {
            // Avoid setting the same conflict value multiple times
            if (i < neighborIdx) {
                atomicExch(&detectConflict[i], 1); // Use atomic to ensure correct setting
            } else {
                atomicExch(&detectConflict[neighborIdx], 1); // Use atomic to ensure correct setting
            }
        }
    }
}