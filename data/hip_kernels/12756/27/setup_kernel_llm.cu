#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState *state, unsigned long seed)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Random state initialization with a unique seed for each thread
    hiprand_init(seed + i, 0, 0, &state[i]);
}