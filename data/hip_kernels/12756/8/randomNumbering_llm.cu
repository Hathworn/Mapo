#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomNumbering (hiprandState* globalState, int *degreeCount, int n, int limit) {
    // Calculate global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if (i < n) { 
        // Use local state for random number generation
        hiprandState localState = globalState[i];
        
        // Generate uniform random number
        float RANDOM = hiprand_uniform(&localState);
        
        // Store updated state back
        globalState[i] = localState;
        
        // Calculate randomized degree
        RANDOM = RANDOM * limit + 1;
        
        // Store result in degree count array
        degreeCount[i] = (int)RANDOM;
    }
}