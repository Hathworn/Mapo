#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Directly manipulate hiprandStatePtr for reduced pointer arithmetic
    hiprandState *statePtr = &state[i];
    
    // Initialize the random state
    hiprand_init(seed, i, 0, statePtr);
}