#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sum = 1;

__global__ void degreeCalc(int *array) {
    // Utilize shared memory for reduced global memory access
    __shared__ int sharedSum;
    if (threadIdx.x == 0) sharedSum = 0;
    __syncthreads();

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < 1000000) {
        atomicAdd(&sharedSum, array[i]);
    }
    __syncthreads();

    // Update global sum from shared sum
    if (threadIdx.x == 0) atomicAdd(&sum, sharedSum);
}

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        int start = vertexArray[i];
        int stop = (i == n - 1) ? 2 * m : vertexArray[i + 1];
        int diff = stop - start;
        degreeCount[i] = diff;
    }
}