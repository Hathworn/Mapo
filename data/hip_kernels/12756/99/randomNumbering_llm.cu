#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomNumbering(hiprandState* globalState, int* degreeCount, int n, int limit) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) { // Bounds check to prevent accessing invalid memory
        hiprandState localState = globalState[i]; // Load state
        float RANDOM = hiprand_uniform(&localState);
        RANDOM = RANDOM * limit + (1 - RANDOM); // Combine limit and addition
        degreeCount[i] = (int)RANDOM;

        globalState[i] = localState; // Store state
    }
}