#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void degreeCalc(int *array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory to reduce global memory accesses
    __shared__ int sharedSum;
    
    // Initialize shared sum for each block
    if (threadIdx.x == 0) {
        sharedSum = 0;
    }
    __syncthreads();

    // Only perform addition if index is valid
    if (i < 1000000) {
        atomicAdd(&sharedSum, array[i]);
    }
    __syncthreads();

    // Use atomic operation to update global sum
    if (threadIdx.x == 0) {
        atomicAdd(&sum, sharedSum);
    }
}

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        
        // Directly assign the difference to avoid variable usage
        degreeCount[i] = vertexArray[i + 1] - vertexArray[i];
    }
}