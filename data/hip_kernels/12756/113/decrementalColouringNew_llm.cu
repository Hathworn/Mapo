#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void decrementalColouringNew(int *vertexArray, int *neighbourArray, int n, int m, int *decrementalArray, int size) {

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Early exit for threads outside array bounds
    if (i >= size) {
        return;
    }

    int me = decrementalArray[i];
    int you = decrementalArray[i ^ 1]; // Use XOR to simplify index calculation for 'you'

    int startStart = vertexArray[me - 1];
    int startStop = vertexArray[me];

    // Iterate with a while loop for potential early exit
    int j = startStart;
    while (j < startStop) {
        if (neighbourArray[j] == you) {
            neighbourArray[j] = 0; // Mark the edge as deleted
            break; // Exit once the edge is found and deleted
        }
        j++;
    }
}