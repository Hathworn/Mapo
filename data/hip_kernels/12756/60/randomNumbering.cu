#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomNumbering (hiprandState* globalState, int *degreeCount, int n, int limit){

int i= blockDim.x * blockIdx.x + threadIdx.x;

hiprandState localState = globalState[i];
float RANDOM = hiprand_uniform( &localState );
globalState[i] = localState;

RANDOM *= (limit - 1 + 0.999999);
RANDOM += 1;

degreeCount[i] = (int) RANDOM;
}