#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomNumbering(hiprandState* globalState, int* degreeCount, int n, int limit) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if thread index is within bounds
    if (i < n) {
        hiprandState localState = globalState[i];
        float RANDOM = hiprand_uniform(&localState);
        globalState[i] = localState;

        // Use integer arithmetic for precision
        int randomValue = (int)(RANDOM * limit);
        degreeCount[i] = randomValue + 1;
    }
}