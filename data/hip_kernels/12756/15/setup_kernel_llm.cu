#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState *state, unsigned long seed) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Launch multiple initialization threads to improve efficiency.
    hiprand_init(seed, i, 0, &state[i]);
}