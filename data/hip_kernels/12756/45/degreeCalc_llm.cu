#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void degreeCalc(int *array) {
    // Efficiently calculate sum using atomic operation
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < 1000000) {
        atomicAdd(&sum, array[i]);
    }
}

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        // Use shared memory for faster access
        extern __shared__ int sharedVertexArray[];

        // Load data into shared memory for coalesced access
        sharedVertexArray[i] = vertexArray[i];

        __syncthreads();

        int start = sharedVertexArray[i];
        int stop = (i == n - 1) ? 2 * m : vertexArray[i + 1];
        
        // Store the difference directly to degreeCount
        degreeCount[i] = stop - start;
    }
}