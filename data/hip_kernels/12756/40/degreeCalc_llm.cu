#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sum = 1;  

__global__ void degreeCalc(int *array) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < 1000000) {
        atomicAdd(&sum, array[i]);  // Use atomicAdd for thread-safe summation
    }
    // Uncomment for debugging output if needed
    // if (i == 999999) {
    //     printf("%d", sum);
    // }
}

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (i < n) {
        int start = vertexArray[i];
        int stop;
        
        if (i == n - 1) {
            stop = 2 * m;
        } else {
            stop = vertexArray[i + 1];
        }
        
        degreeCount[i] = stop - start;  // Calculate degree directly
    }
}