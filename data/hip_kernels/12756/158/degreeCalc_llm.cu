#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256

__global__ void degreeCalc(int *vertexArray, int *neighbourArray, int *degreeCount, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Check within bounds for the work item
    if (i < n) {
        int start = vertexArray[i];
        int stop = vertexArray[i + 1];
        degreeCount[i] = stop - start;  // Calculate degree
    }
}