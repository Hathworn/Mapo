#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomNumbering(hiprandState* globalState, int *degreeCount, int n, int limit) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= n) return; // Ensure i is within bounds

    hiprandState localState = globalState[i];
    float RANDOM = hiprand_uniform(&localState);

    RANDOM = RANDOM * (limit - 1) + 1.0f; // Optimize calculation

    degreeCount[i] = __float2int_rn(RANDOM); // More accurate float to int conversion
    globalState[i] = localState;
}