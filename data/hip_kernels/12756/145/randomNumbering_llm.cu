#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomNumbering (hiprandState* globalState, int *degreeCount, int n, int limit){

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if (i < n) {
        hiprandState localState = globalState[i];
        float RANDOM = hiprand_uniform(&localState);
        
        // Update global state after random number generation
        globalState[i] = localState;

        // Scale and shift the random number
        RANDOM *= limit;
        degreeCount[i] = (int) RANDOM + 1;
    }
}