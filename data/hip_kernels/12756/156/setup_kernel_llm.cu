#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function
__global__ void setup_kernel (hiprandState * state, unsigned long seed ){
    // Use shared memory for block-wise operations (if applicable to your use case)
    __shared__ hiprandState localState[256]; // Adjust size based on block dimension

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Each thread initializes its own state in shared memory
    hiprand_init(seed, i, 0, &localState[threadIdx.x]);

    // Ensure all threads have finished initialization before copying to global state
    __syncthreads();

    // Copy initialized state to global memory
    state[i] = localState[threadIdx.x];
}