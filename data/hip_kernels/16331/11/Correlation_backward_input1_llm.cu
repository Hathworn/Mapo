#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Correlation_backward_input1(int item, float *gradInput1, int nInputChannels, int inputHeight, int inputWidth, float *gradOutput, int nOutputChannels, int outputHeight, int outputWidth, float *rInput2, int pad_size, int kernel_size, int max_displacement, int stride1, int stride2)
{
    // Calculate thread coordinates
    int n = item;
    int y = blockIdx.x * stride1 + pad_size;
    int x = blockIdx.y * stride1 + pad_size;
    int c = blockIdx.z;
    int tch_off = threadIdx.x;

    // Pre-compute commonly used values to minimize calculations in the loop
    int kernel_rad = (kernel_size - 1) / 2;
    int displacement_rad = max_displacement / stride2;
    int displacement_size = 2 * displacement_rad + 1;
    int pInputWidth = inputWidth + 2 * pad_size;
    int pdimc = nInputChannels;
    float nelems = kernel_size * kernel_size * nInputChannels;

    // Calculate bounds, optimized initialization
    int xmin = max(0, (x - kernel_rad - max_displacement) / stride1);
    int ymin = max(0, (y - kernel_rad - max_displacement) / stride1);
    int xmax = min(outputWidth-1, (x + kernel_rad - max_displacement) / stride1);
    int ymax = min(outputHeight-1, (y + kernel_rad - max_displacement) / stride1);

    if (xmax < xmin || ymax < ymin) return;

    // Declare shared memory for partial product sums
    __shared__ float prod_sum[CUDA_NUM_THREADS];
    prod_sum[tch_off] = 0;  // Reset shared memory for each thread

    // Loop over output channels
    for (int tc = tch_off; tc < nOutputChannels; tc += CUDA_NUM_THREADS) {

        // Calculate i2 and j2, reuse common computations
        int i2 = (tc % displacement_size - displacement_rad) * stride2;
        int j2 = (tc / displacement_size - displacement_rad) * stride2;
        
        int indx2 = n * pInputWidth * (inputHeight + 2 * pad_size) * nInputChannels + (y + j2) * pInputWidth * pdimc + (x + i2) * pdimc + c;
        float val2 = rInput2[indx2];

        // Loop over the relevant grid region and compute product sum
        for (int j = ymin; j <= ymax; ++j) {
            for (int i = xmin; i <= xmax; ++i) {
                int tindx = n * nOutputChannels * outputHeight * outputWidth + tc * outputHeight * outputWidth + j * outputWidth + i;
                prod_sum[tch_off] += gradOutput[tindx] * val2;
            }
        }
    }
    
    // Synchronize and reduce results
    __syncthreads();
    if(tch_off == 0) {
        float reduce_sum = 0;
        for(int idx = 0; idx < CUDA_NUM_THREADS; idx++) {
            reduce_sum += prod_sum[idx];
        }
        const int indx1 = n * nInputChannels * inputHeight * inputWidth + c * inputHeight * inputWidth + (y - pad_size) * inputWidth + (x - pad_size);
        gradInput1[indx1] = reduce_sum / nelems;
    }
}