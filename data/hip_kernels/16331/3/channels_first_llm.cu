#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void channels_first(float* input, float* rinput, int channels, int height, int width, int pad_size)
{
    // Use shared memory for faster access
    __shared__ float tile[THREADS_PER_BLOCK];

    int n = blockIdx.x;
    int y = blockIdx.y;
    int x = blockIdx.z;

    int ch_off = threadIdx.x;

    int dimcyx = channels * height * width;
    int dimyx = height * width;

    int p_dimx = (width + 2 * pad_size);
    int p_dimy = (height + 2 * pad_size);
    int p_dimyxc = channels * p_dimy * p_dimx;
    int p_dimxc = p_dimx * channels;

    int input_idx, rinput_idx;

    // Stride over channels
    for (int c = ch_off; c < channels; c += THREADS_PER_BLOCK) {
        input_idx = n * dimcyx + c * dimyx + y * width + x;
        // Load into shared memory
        tile[threadIdx.x] = input[input_idx];
        __syncthreads();  // Ensure all threads have loaded their values
        rinput_idx = n * p_dimyxc + (y + pad_size) * p_dimxc + (x + pad_size) * channels + c;
        // Store from shared memory
        rinput[rinput_idx] = tile[threadIdx.x];
        __syncthreads();  // Ensure all threads have read their values
    }
}