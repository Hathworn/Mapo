#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Correlation_backward_input1(int item, float *gradInput1, int nInputChannels, int inputHeight, int inputWidth, float *gradOutput, int nOutputChannels, int outputHeight, int outputWidth, float *rInput2, int pad_size, int kernel_size, int max_displacement, int stride1, int stride2)
{
    // Calculate pixel indices with padding offset for the batch item
    int n = item;
    int y = blockIdx.x * stride1 + pad_size;
    int x = blockIdx.y * stride1 + pad_size;
    int c = blockIdx.z;
    int tch_off = threadIdx.x;

    // Precompute constants
    int kernel_rad = (kernel_size - 1) / 2;
    int displacement_rad = max_displacement / stride2;
    int displacement_size = 2 * displacement_rad + 1;
    
    // Compute bounds for processing
    int xmin = (x - kernel_rad - max_displacement) / stride1;
    int ymin = (y - kernel_rad - max_displacement) / stride1;
    int xmax = (x + kernel_rad - max_displacement) / stride1;
    int ymax = (y + kernel_rad - max_displacement) / stride1;

    // Return early if out of bounds
    if (xmax < 0 || ymax < 0 || xmin >= outputWidth || ymin >= outputHeight) {
        return;
    }
    if (xmin > xmax || ymin > ymax) {
        return;
    }

    // Clamp bounds within valid range
    xmin = max(0, xmin);
    xmax = min(outputWidth - 1, xmax);
    ymin = max(0, ymin);
    ymax = min(outputHeight - 1, ymax);

    // Precompute dimensional offsets
    int pInputWidth = inputWidth + 2 * pad_size;
    int pInputHeight = inputHeight + 2 * pad_size;
    int pdimyxc = pInputHeight * pInputWidth * nInputChannels;
    int pdimxc = pInputWidth * nInputChannels;
    int pdimc = nInputChannels;
    int tdimcyx = nOutputChannels * outputHeight * outputWidth;
    int tdimyx = outputHeight * outputWidth;
    int tdimx = outputWidth;

    // Compute total elements of kernel window
    float nelems = kernel_size * kernel_size * nInputChannels;

    // Shared memory allocation for warp-wide sum
    __shared__ float prod_sum[CUDA_NUM_THREADS];
    prod_sum[tch_off] = 0;

    // Accumulate results across channels within a block
    for (int tc = tch_off; tc < nOutputChannels; tc += CUDA_NUM_THREADS) {
        int i2 = (tc % displacement_size - displacement_rad) * stride2;
        int j2 = (tc / displacement_size - displacement_rad) * stride2;
        int indx2 = n * pdimyxc + (y + j2) * pdimxc + (x + i2) * pdimc + c;

        // Load co-located value from rInput2
        float val2 = rInput2[indx2];

        // Accumulate correlation contributions within specified bounds
        for (int j = ymin; j <= ymax; ++j) {
            for (int i = xmin; i <= xmax; ++i) {
                int tindx = n * tdimcyx + tc * tdimyx + j * tdimx + i;
                prod_sum[tch_off] += gradOutput[tindx] * val2;
            }
        }
    }
    __syncthreads();

    // Accumulate results across threads
    if (tch_off == 0) {
        float reduce_sum = 0;
        for (int idx = 0; idx < CUDA_NUM_THREADS; idx++) {
            reduce_sum += prod_sum[idx];
        }
        const int indx1 = n * pdimyxc + c * (pInputHeight * inputWidth) + (y - pad_size) * inputWidth + (x - pad_size);
        gradInput1[indx1] = reduce_sum / nelems;
    }
}