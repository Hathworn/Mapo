#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256 // Ensure this is defined

__global__ void channels_first(float* input, float* rinput, int channels, int height, int width, int pad_size) {
    // n (batch size), y (height), x (width)
    int n = blockIdx.x;
    int y = blockIdx.y;
    int x = blockIdx.z;

    int ch_off = threadIdx.x;

    // Pre-compute reused variables
    int dimcyx = channels * height * width;
    int dimyx = height * width;
    int p_dimx = (width + 2 * pad_size);
    int p_dimy = (height + 2 * pad_size);
    int p_dimyxc = channels * p_dimy * p_dimx;
    int p_dimxc = p_dimx * channels;

    // Use shared memory for block-level data reuse
    __shared__ float sharedInput[THREADS_PER_BLOCK];

    for (int c = ch_off; c < channels; c += THREADS_PER_BLOCK) {
        // Efficient memory access with coalesced reads
        sharedInput[threadIdx.x] = input[n * dimcyx + c * dimyx + y * width + x];

        // Wait for all threads to load necessary data
        __syncthreads();

        // Store results utilizing shared memory for optimal write pattern
        rinput[n * p_dimyxc + (y + pad_size) * p_dimxc + (x + pad_size) * channels + c] = sharedInput[threadIdx.x];

        // Ensure all writes are completed before next iteration
        __syncthreads();
    }
}