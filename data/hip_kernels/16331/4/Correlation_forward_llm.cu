#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Correlation_forward(float *output, int nOutputChannels, int outputHeight, int outputWidth, float *rInput1, int nInputChannels, int inputHeight, int inputWidth, float *rInput2, int pad_size, int kernel_size, int max_displacement, int stride1, int stride2) {

    int pInputWidth = inputWidth + 2 * pad_size;
    int pInputHeight = inputHeight + 2 * pad_size;

    int kernel_rad = (kernel_size - 1) / 2;
    int displacement_rad = max_displacement / stride2;
    int displacement_size = 2 * displacement_rad + 1;

    int n  = blockIdx.x;
    int base_y1 = blockIdx.y * stride1 + max_displacement + kernel_rad;
    int base_x1 = blockIdx.z * stride1 + max_displacement + kernel_rad;
    int c = threadIdx.x;

    int pdimyxc = pInputHeight * pInputWidth * nInputChannels;
    int pdimxc = pInputWidth * nInputChannels;
    int pdimc = nInputChannels;

    int tdimcyx = nOutputChannels * outputHeight * outputWidth;
    int tdimyx = outputHeight * outputWidth;
    int tdimx = outputWidth;

    float nelems = kernel_size * kernel_size * pdimc;

    __shared__ float prod_sum[BLOCK_SIZE_X];
    prod_sum[c] = 0.0f;  // Initialize shared memory

    // Parallelize over the elements in the displacement window
    for (int tj = -displacement_rad; tj <= displacement_rad; ++tj) {
        int y2 = base_y1 + tj * stride2;
        for (int ti = -displacement_rad; ti <= displacement_rad; ++ti) {
            int x2 = base_x1 + ti * stride2;

            float sum = 0.0f;
            // Parallelize over the kernel region
            for (int j = -kernel_rad; j <= kernel_rad; ++j) {
                for (int i = -kernel_rad; i <= kernel_rad; ++i) {
                    int indx1_base = n * pdimyxc + (base_y1 + j) * pdimxc + (base_x1 + i) * pdimc;
                    int indx2_base = n * pdimyxc + (y2 + j) * pdimxc + (x2 + i) * pdimc;

                    for (int ch = c; ch < pdimc; ch += BLOCK_SIZE_X) {
                        sum += rInput1[indx1_base + ch] * rInput2[indx2_base + ch];
                    }
                }
            }

            prod_sum[c] = sum;

            // Accumulate with reduction
            __syncthreads();
            if (c == 0) {
                float reduce_sum = 0;
                for (int index = 0; index < BLOCK_SIZE_X; ++index) {
                    reduce_sum += prod_sum[index];
                }
                int tc = (tj + displacement_rad) * displacement_size + (ti + displacement_rad);
                const int tindx = n * tdimcyx + tc * tdimyx + blockIdx.y * tdimx + blockIdx.z;
                output[tindx] = reduce_sum / nelems;
            }
            __syncthreads();
        }
    }
}
```
