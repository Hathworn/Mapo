#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void colNorm(float* X, float* v, unsigned int size, unsigned int n) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Iterate through elements with stride
    for (unsigned int i = idx; i < size; i += blockDim.x * gridDim.x) {
        // Compute column index
        unsigned int col = i % n;
        // Normalize element by corresponding column value
        X[i] /= v[col];
    }
}