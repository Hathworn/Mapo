#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void dual(float* err, float* Y, float* X, float* Z, unsigned int size)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    float err_temp = 0.0;  // Use local variable to accumulate errors

    for (unsigned int i = idx; i < size; i += stride) {
        float temp = X[i] - Z[i];
        Y[i] += temp;
        err_temp += temp * temp;  // Accumulate errors locally
    }

    atomicAdd(&err[idx], err_temp);  // Atomic add to accumulate global errors
}