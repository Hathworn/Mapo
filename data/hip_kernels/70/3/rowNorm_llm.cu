#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void rowNorm(float* X, float* v, unsigned int size, unsigned int n)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    // Utilize more registers efficiently by removing the cast to int
    for (unsigned long int i = idx; i < size; i += stride) {
        unsigned int row = i / n; // Calculate row index using division
        X[i] /= v[row]; // Divide by corresponding row normalization factor
    }
}