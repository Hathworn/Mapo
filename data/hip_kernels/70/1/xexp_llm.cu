#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void xexp(float* X, float* C, float* Y, float* Z, unsigned int size)
{
    // Calculate thread's global index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride to cover all elements
    const unsigned int stride = blockDim.x * gridDim.x;
    
    // Unroll loop to improve performance
    for (unsigned long int i = idx; i < size; i += stride * 4) {
        if (i < size) X[i] = Z[i] * __expf(C[i] - Y[i]);
        if (i + stride < size) X[i + stride] = Z[i + stride] * __expf(C[i + stride] - Y[i + stride]);
        if (i + 2 * stride < size) X[i + 2 * stride] = Z[i + 2 * stride] * __expf(C[i + 2 * stride] - Y[i + 2 * stride]);
        if (i + 3 * stride < size) X[i + 3 * stride] = Z[i + 3 * stride] * __expf(C[i + 3 * stride] - Y[i + 3 * stride]);
    }
}