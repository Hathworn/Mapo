#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void rowNorm_a(float* X, float* v, float* a, unsigned int size, unsigned int n)
{
    // Calculate row once outside loop
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned long int i = idx; i < size; i += stride) {
        unsigned int row = i / n; // Use unsigned int for row to avoid unnecessary casting
        float factor = v[row] * a[row]; // Calculate multiplication once
        X[i] /= factor; // Use the pre-calculated factor
    }
}