#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/


__global__ void colNorm_b(float* X, float* v, float* b, unsigned int size, unsigned int n) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned long int i = idx; i < size; i += stride) {
        unsigned int col = i % n; // Simplified calculation of column index
        float normalizationFactor = v[col] * b[col]; // Precompute normalization factor
        X[i] /= normalizationFactor; // Use precomputed factor for division
    }
}