#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void reduce(float *g_idata, float *g_odata, unsigned int n)
{
    extern __shared__ float sdata[];

    // Initial read of data from global memory to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    unsigned int gridSize = blockDim.x * 2 * gridDim.x;

    float mySum = 0;

    // Loop through grid using stride to compute partial reduction
    while (i < n) {
        mySum += g_idata[i];
        if (i + blockDim.x < n) mySum += g_idata[i + blockDim.x];
        i += gridSize;
    }

    sdata[tid] = mySum; 
    __syncthreads();

    // Reduction in shared memory using unrolled loops
    if (blockDim.x >= 1024) { if (tid < 512) { sdata[tid] += sdata[tid + 512]; } __syncthreads(); } 
    if (blockDim.x >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); } 
    if (blockDim.x >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); } 
    if (blockDim.x >= 128) { if (tid <  64) { sdata[tid] += sdata[tid +  64]; } __syncthreads(); }

    // Final warp-level reduction within 64 elements, avoiding bank conflicts
    if (tid < 32) {
        volatile float* smem = sdata;
        if (blockDim.x >= 64) smem[tid] += smem[tid + 32];
        if (blockDim.x >= 32) smem[tid] += smem[tid + 16];
        if (blockDim.x >= 16) smem[tid] += smem[tid +  8];
        if (blockDim.x >=  8) smem[tid] += smem[tid +  4];
        if (blockDim.x >=  4) smem[tid] += smem[tid +  2];
        if (blockDim.x >=  2) smem[tid] += smem[tid +  1];
    }

    // Output final result for this block
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}