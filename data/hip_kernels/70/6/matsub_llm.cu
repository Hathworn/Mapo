#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void matsub(float* X, float* Y, unsigned int size)
{
    // Calculate global thread ID
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop unrolling optimization
    int stride = blockDim.x * gridDim.x;
    for (unsigned int i = idx; i < size; i += stride) {
        X[i] -= Y[i];
    }
}