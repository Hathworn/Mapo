#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/***********************************************************
By Huahua Wang, the University of Minnesota, twin cities
***********************************************************/

__global__ void zexp(float* Z, float* X, float* Y, unsigned int size)
{
    // Calculate global index for the thread
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid-stride loop for better memory access and load balancing
    for (unsigned int i = idx; i < size; i += blockDim.x * gridDim.x) {
        Z[i] = X[i] * __expf(Y[i]); // Utilize fast math intrinsic for exp
    }
}