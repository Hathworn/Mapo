#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void doubleToFloat(const double* __restrict__ input, float* __restrict__ output, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Use shared memory for coalesced access if necessary
    if (i < numElements)
    {
        output[i] = __double2float_rd(input[i]); // Use intrinsic for type conversion
    }
}