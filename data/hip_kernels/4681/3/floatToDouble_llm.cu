#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void floatToDouble(float* input, double* output, int numElements)
{
    // Use shared memory to reduce global memory access latency
    extern __shared__ float sharedInput[];

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;

    // Load input to shared memory
    if (i < numElements) {
        sharedInput[tid] = input[i];
    }
    __syncthreads();

    // Perform computation using shared memory
    if (i < numElements) {
        output[i] = static_cast<double>(sharedInput[tid]);
    }
}