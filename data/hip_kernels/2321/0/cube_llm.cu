#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cube(float * d_out, float * d_in) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Change to use block index for unique global index
    float f = d_in[idx];
    d_out[idx] = f * f * f;
}