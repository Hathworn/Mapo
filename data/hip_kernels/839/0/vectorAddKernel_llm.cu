#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAddKernel(float* deviceA, float* deviceB, float* deviceResult, int n) {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use if condition to avoid accessing out of bounds
    if (i < n) {
        deviceResult[i] = deviceA[i] + deviceB[i];
    }
}