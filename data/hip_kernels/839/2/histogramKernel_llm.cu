#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogramKernel(unsigned char* image, long img_size, unsigned int* histogram, int hist_size) {
    // Define shared memory for local histogram
    extern __shared__ unsigned int local_hist[];

    // Initialize shared memory to zero
    int tid = threadIdx.x;
    int global_index = blockIdx.x * blockDim.x + tid;
    if (tid < hist_size) {
        local_hist[tid] = 0;
    }
    __syncthreads();

    // Aggregate data into local histogram using a stride loop
    for (long i = global_index; i < img_size; i += blockDim.x * gridDim.x) {
        atomicAdd(&local_hist[image[i]], 1);
    }
    __syncthreads();

    // Combine local histograms into global histogram using atomic operations
    if (tid < hist_size) {
        atomicAdd(&histogram[tid], local_hist[tid]);
    }
}