#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized histogram kernel
__global__ void histogramKernel(unsigned char* image, long img_size, unsigned int* histogram, int hist_size) {
    extern __shared__ unsigned int temp_hist[]; // Shared memory for partial histograms
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Initialize shared memory
    for (int i = threadIdx.x; i < hist_size; i += blockDim.x)
        temp_hist[i] = 0;
    __syncthreads();

    // Populate shared memory histogram
    for (int i = tid; i < img_size; i += stride)
        atomicAdd(&temp_hist[image[i]], 1);
    __syncthreads();

    // Combine partial histograms into global memory
    for (int i = threadIdx.x; i < hist_size; i += blockDim.x)
        atomicAdd(&histogram[i], temp_hist[i]);
}