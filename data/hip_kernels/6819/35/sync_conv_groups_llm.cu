#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups() {
    __shared__ int dummy; // Use shared memory for better synchronization
    dummy = threadIdx.x;  // Ensure all threads reach this point
    __syncthreads();      // Synchronize all threads in the block
}