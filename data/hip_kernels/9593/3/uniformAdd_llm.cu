#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) {
    __shared__ int uni; // Change type to int for consistency with input arrays
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];

    unsigned int address = __mul24(blockIdx.x, (blockDim.x << 1)) + baseIndex + threadIdx.x;

    __syncthreads();

    // Optimized adds with boundary check and shared memory usage
    if (address < n) {
        g_data[address] += uni;
    }
    if (address + blockDim.x < n) {
        g_data[address + blockDim.x] += uni;
    }
}