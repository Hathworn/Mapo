#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize loading imag part
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

// Optimize global kernel function
__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Use shared memory to reduce global memory accesses
    __shared__ float shared_value;

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        shared_value = value;
    }
    __syncthreads();

    if (i < h && j < w) {
        const int pos = i * w + j;
        image[pos] = shared_value; // Use shared memory value
    }
}