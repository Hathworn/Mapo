#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate the unique thread index
    int tid = blockIdx.y * blockDim.x * blockDim.y * gridDim.x + blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    int totalThreads = w * h;

    // Unroll loop for efficiency
    for (int idx = tid; idx < totalThreads; idx += blockDim.x * blockDim.y * gridDim.x * gridDim.y) {
        int i = idx / w;
        int j = idx % w;
        if (i >= h || j >= w) return;

        const int pos = i * s + j;

        float scale = normalization_factor[pos];
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

        image[pos] *= invScale;
    }
}