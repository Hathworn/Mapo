#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Early exit if out of bounds
    if (i >= h || j >= w) return;

    // Precompute offsets
    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Fetch flow values
    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Calculate coordinates with pre-added offsets
    float cx = fmaf(u_, time_scale, j + 1.0f);
    float cy = fmaf(v_, time_scale, i + 1.0f);

    // Round to nearest integer pixel location
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Fetch source value
    float value = src[image_row_offset + j];

    // Atomic add only within bounds
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        atomicAdd(&dst[ty * image_stride + tx], value);
    }
}