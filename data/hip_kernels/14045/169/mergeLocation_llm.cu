#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Fetch location to a register for faster access
        short2 loc = loc_[ptidx];

        // Perform computation and directly store in global memory
        float2 scaled_loc = make_float2(loc.x * scale, loc.y * scale);

        x[ptidx] = scaled_loc.x;
        y[ptidx] = scaled_loc.y;
    }
}