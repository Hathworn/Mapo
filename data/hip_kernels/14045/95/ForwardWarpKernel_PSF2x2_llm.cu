#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // Pixel containing bottom left corner
    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);

    // Target pixel integer coordinates
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);

    float value = src[image_row_offset + j];

    // Precompute weights for efficiency
    float dxdy = dx * dy;
    float dx1dy = (1.0f - dx) * dy;
    float dxdy1 = dx * (1.0f - dy);
    float dx1dy1 = (1.0f - dx) * (1.0f - dy);

    // Fill pixels using precomputed weights
    // Bottom right corner
    if (tx < w && tx >= 0 && ty < h && ty >= 0)
    {
        _atomicAdd(dst + ty * image_stride + tx, value * dxdy);
        _atomicAdd(normalization_factor + ty * image_stride + tx, dxdy);
    }

    // Bottom left corner
    tx -= 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0)
    {
        _atomicAdd(dst + ty * image_stride + tx, value * dx1dy);
        _atomicAdd(normalization_factor + ty * image_stride + tx, dx1dy);
    }

    // Upper left corner
    ty -= 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0)
    {
        _atomicAdd(dst + ty * image_stride + tx, value * dx1dy1);
        _atomicAdd(normalization_factor + ty * image_stride + tx, dx1dy1);
    }

    // Upper right corner
    tx += 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0)
    {
        _atomicAdd(dst + ty * image_stride + tx, value * dxdy1);
        _atomicAdd(normalization_factor + ty * image_stride + tx, dxdy1);
    }
}