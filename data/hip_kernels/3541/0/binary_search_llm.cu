#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binary_search(int* a, int* b, bool* c, int sizeofa) // optimized kernel function
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= sizeofa) return; // Avoid out-of-bounds access
    int key = b[index];
    int min = 0, max = sizeofa - 1;
    int mid;
    bool found = false;

    while (min <= max) 
    {
        mid = (min + max) / 2;
        if (key == a[mid]) 
        {
            found = true;
            break; // Key found
        } 
        else if (key < a[mid]) 
        {
            max = mid - 1;
        } 
        else 
        {
            min = mid + 1;
        }
    }

    c[index] = found;

    // Remove print statements for better performance
}