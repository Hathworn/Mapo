#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use __restrict__ qualifiers for pointers to improve memory access efficiency
// Unroll loops when possible to reduce the loop overhead
__global__ void add_vector(int* __restrict__ a, int* __restrict__ b, int* __restrict__ c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {  // Ensure we don't go out of bounds
        c[i] = a[i] + b[i];
    }
}