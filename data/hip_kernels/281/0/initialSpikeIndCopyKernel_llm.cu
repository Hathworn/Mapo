#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Error checking */
#define CUDA_ERROR_CHECK
#define CURAND_ERROR_CHECK
#define CUDA_CALL( err) __cudaCall( err, __FILE__, __LINE__ )
#define CURAND_CALL( err) __curandCall( err, __FILE__, __LINE__)
#define CUDA_CHECK_ERROR()    __cudaCheckError( __FILE__, __LINE__ )

__global__ void initialSpikeIndCopyKernel(unsigned short* pLastSpikeInd, const unsigned int noReal)
{
    // Calculate the global index once and reuse it
    unsigned int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (globalIndex < noReal * noSpikes) // Ensure no boundary overflow
    {
        // Use local variable to store computed index for efficient memory access
        unsigned int spikeNo = globalIndex / noReal;
        pLastSpikeInd[globalIndex] = pLastSpikeInd[spikeNo * noReal];
    }
}