#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initialSpikeIndCopyKernel(unsigned short* pLastSpikeInd, const unsigned int noReal, const unsigned int noSpikes) 
{
    unsigned int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (globalIndex < noReal * noSpikes) {
        // Calculate spike number efficiently
        unsigned int spikeNo = globalIndex / noReal;
        // Efficiently copy spike index
        pLastSpikeInd[globalIndex] = __ldg(&pLastSpikeInd[spikeNo * noReal]);
    }
}