#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    // Use shared memory to reduce global memory access
    __shared__ float shared_data[CU1DBLOCK * 2];
    
    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {
        if (idx == 0) continue;

        // Load data into shared memory
        shared_data[2 * thread_id] = Ar[2 * idx];
        shared_data[2 * thread_id + 1] = Ar[2 * idx + 1];
        __syncthreads();

        float val_x = shared_data[2 * thread_id];
        float val_y = shared_data[2 * thread_id + 1];
        
        float ret = val_x * val_x + val_y * val_y;
        if (use_power) {
            Aw[idx] = ret;
        } else {
            Aw[idx] = sqrtf(ret);
        }
        __syncthreads();
    }

    // Special case handling by the first thread in block
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        if (use_power) {
            Aw[0] = real * real;
            Aw[half_length] = im * im;
        } else {
            Aw[0] = fabs(real);
            Aw[half_length] = fabs(im);
        }
    }
}