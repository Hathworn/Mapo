#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_linear_and_quadratic_terms_kernel( int32_t n, float old_num_frames, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    float cur_weight = *cur_tot_weight;
    float new_num_frames = old_num_frames + cur_weight;
    float prior_scale_change = 1.0f;

    if (max_count != 0.0f) {
        float old_prior_scale = max(old_num_frames, (float)max_count) / max_count;
        float new_prior_scale = max(new_num_frames, (float)max_count) / max_count;
        prior_scale_change += new_prior_scale - old_prior_scale;
    }

    // Calculate global thread index
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride = blockDim.x * gridDim.x;

    // Use stride loop to improve cache usage and prevent idle threads
    for (int32_t i = idx; i < n; i += stride) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        quadratic[diag_idx] += prior_scale_change;
    }

    // Use a single thread to update the first element of linear array
    if (idx == 0) {
        linear[0] += prior_offset * prior_scale_change;
    }
}