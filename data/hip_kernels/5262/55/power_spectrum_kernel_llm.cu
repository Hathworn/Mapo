#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    const float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    
    // Use shared memory for efficiency if possible (depending on size)
    float2 val;
    float ret;
    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {  // Use block dimension for increment
        if (idx == 0) continue;  // Ignore special case in loop

        val = reinterpret_cast<const float2 *>(Ar)[idx];
        ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret);  // Use ternary operator for simplicity
    }

    // Handle special cases at the start and end
    if (thread_id == 0) {
        float real = Ar[0];
        float im = Ar[row_length];
        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}