#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Compute global thread index for row and column
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process only if within matrix dimensions
    if (row < num_rows && col < num_cols) {
        float f = feats[row * ldf + col];
        // Squaring the element
        feats_sq[row * lds + col] = f * f;
    }
}