#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256
#define CHUNK_SIZE 16

int randNotSeeded = 1;

// Inline the insertion sort for better performance
__device__ void insertionSort(int *array, int a, int b)
{
    int current;
    for (int i = a + 1; i < b; i++)
    {
        current = array[i];
        int j = i - 1;
        // Combine condition checks to reduce branch instructions
        while (j >= a && current < array[j])
        {
            array[j + 1] = array[j];
            j--;
        }
        array[j + 1] = current;
    }
}

__global__ void gpu_sort(int *d_array, int size, int chunkSize)
{
    // Pre-calculate index to avoid recalculating within the kernel
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int a = idx * chunkSize;
    if (a >= size) return;

    int b = min(a + chunkSize, size); // Use min to directly compute the boundary

    insertionSort(d_array, a, b);
}