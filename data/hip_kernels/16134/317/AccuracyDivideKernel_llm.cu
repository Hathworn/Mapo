#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Ensure single thread performs the division to prevent race conditions
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}