#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const float PI = 4*atan(1);

__global__ void jacobiMethod(float* grid, float* potential, int sizeX, int sizeY, float scale, int noIters, float tolerance) {

    extern __shared__ float sharedMem[];

    int threadIdX = threadIdx.x;
    int threadIdY = threadIdx.y;

    int bOx = blockIdx.x * blockDim.x;
    int bOy = blockIdx.y * blockDim.y;

    int effBlockSizeX = blockDim.x + 2 * noIters;
    int effBlockSizeY = blockDim.y + 2 * noIters;

    int totalSize = sizeX * sizeY;
    int sharedMemSize = effBlockSizeX * effBlockSizeY;

    // Optimize data copying with boundary checks
    for(int i = threadIdX; i < effBlockSizeX; i += blockDim.x) {
        for (int j = threadIdY; j < effBlockSizeY; j += blockDim.y) {
            int currElemSM = i * effBlockSizeX + j;
            int currElemMain = (i - noIters + bOy) * sizeX + (j - noIters + bOx);
            if (currElemMain >= 0 && currElemMain < totalSize) {
                sharedMem[currElemSM] = grid[currElemMain];
                sharedMem[currElemSM + sharedMemSize] = potential[currElemMain];
            } else {
                sharedMem[currElemSM] = 0;
                sharedMem[currElemSM + sharedMemSize] = 0;
            }
            sharedMem[currElemSM + 2 * sharedMemSize] = 0;
        }
    }
    __syncthreads();

    for(int k = 0; k < noIters; k++) {
        for(int i = threadIdX; i < effBlockSizeX; i += blockDim.x) {
            for(int j = threadIdY; j < effBlockSizeY; j += blockDim.y) {
                int currPos = i * effBlockSizeX + j + 2 * sharedMemSize;
                sharedMem[currPos] = 0;
                // Aggregate neighbors using boundary checks
                if(i > 0) sharedMem[currPos] += (sharedMem[currPos - effBlockSizeY - sharedMemSize] / 4);
                if(i < effBlockSizeX - 1) sharedMem[currPos] += (sharedMem[currPos + effBlockSizeY - sharedMemSize] / 4);
                if(j > 0) sharedMem[currPos] += (sharedMem[currPos - 1 - sharedMemSize] / 4);
                if(j < effBlockSizeY - 1) sharedMem[currPos] += (sharedMem[currPos + 1 - sharedMemSize] / 4);

                if(i < effBlockSizeX - 1 && j < effBlockSizeY - 1) {
                    sharedMem[currPos] += (scale * scale / 4 * sharedMem[currPos - 2 * sharedMemSize]);
                }
            }
        }
        __syncthreads();

        for (int i = threadIdX; i < effBlockSizeX; i += blockDim.x) {
            for (int j = threadIdY; j < effBlockSizeY; j += blockDim.y) {
                int currPos = i * effBlockSizeX + j + 2 * sharedMemSize;
                sharedMem[currPos - sharedMemSize] = sharedMem[currPos];
            }
        }
        __syncthreads();
    }

    // Optimize copying back with boundary checks
    for (int i = threadIdX; i < effBlockSizeX; i += blockDim.x) {
        for (int j = threadIdY; j < effBlockSizeY; j += blockDim.y) {
            if (i >= noIters && j >= noIters && i < effBlockSizeX - noIters && j < effBlockSizeY - noIters) {
                int currElemSM = i * effBlockSizeX + j;
                int currElemMain = (i - noIters + bOy) * sizeX + (j - noIters + bOx);
                if (currElemMain > 0 && currElemMain < totalSize) {
                    potential[currElemMain] = sharedMem[currElemSM + 2 * sharedMemSize];
                }
            }
        }
    }
}