#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    // Calculate a unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop to handle out-of-bound elements
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < N; i += stride) {
        a[i] = a[i] * a[i];
    }
}