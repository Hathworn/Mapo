#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lookupTableCuda(const uint8_t* __restrict__ in, uint32_t rowSizeIn, uint8_t* __restrict__ out, uint32_t rowSizeOut, uint32_t width, uint32_t height, const uint8_t* __restrict__ table) {
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure threads operate within bounds
    if (x < width && y < height) {
        // Optimized memory access with restricted pointers and removed recalculations
        const uint32_t inputIndex = y * rowSizeIn + x;
        const uint32_t outputIndex = y * rowSizeOut + x;
        out[outputIndex] = table[in[inputIndex]];
    }
}