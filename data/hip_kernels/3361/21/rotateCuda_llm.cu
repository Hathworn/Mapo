#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rotateCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, float inXStart, float inYStart, uint32_t width, uint32_t height, float cosAngle, float sinAngle) {
    uint32_t outX = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t outY = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if current thread is processing a valid pixel
    if (outX < width && outY < height) {
        // Calculate transformed input coordinates using rotation
        const float exactInX = inXStart + cosAngle * outX + sinAngle * outY;
        const float exactInY = inYStart - sinAngle * outX + cosAngle * outY;

        const int32_t inX = static_cast<int32_t>(exactInX);
        const int32_t inY = static_cast<int32_t>(exactInY);

        // Calculate output pixel memory offset
        uint8_t* outputPixel = out + outY * rowSizeOut + outX;

        // Check if input coordinates are within valid image bounds
        if (inX < 0 || inX >= width - 1 || inY < 0 || inY >= height - 1) {
            *outputPixel = 0; // Set out-of-bounds pixels to 0
        } else {
            // Calculate input pixel memory offset
            const uint8_t* inputPixel = in + inY * rowSizeIn + inX;

            // Perform bilinear interpolation
            const float probX = exactInX - inX;
            const float probY = exactInY - inY;
            const float mean = *inputPixel * (1 - probX) * (1 - probY) +
                               *(inputPixel + 1) * probX * (1 - probY) +
                               *(inputPixel + rowSizeIn) * (1 - probX) * probY +
                               *(inputPixel + rowSizeIn + 1) * probX * probY + 0.5f;

            *outputPixel = static_cast<uint8_t>(mean);
        }
    }
}