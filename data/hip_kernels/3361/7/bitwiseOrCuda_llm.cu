#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bitwiseOrCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
    // Calculate global thread ID
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure thread is within bounds of the image
    if (x < width && y < height) {
        const uint32_t idIn1 = y * rowSizeIn1 + x;
        const uint32_t idIn2 = y * rowSizeIn2 + x;
        const uint32_t idOut = y * rowSizeOut + x;
        
        // Perform bitwise OR operation
        out[idOut] = in1[idIn1] | in2[idIn2];
    }
}