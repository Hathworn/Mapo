#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void absoluteDifferenceCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height) 
{
    // Calculate single index for x and y to avoid recomputation
    const uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx < width && idy < height) 
    {
        // Use a single calculation for offsets
        const uint32_t in1Offset = idy * rowSizeIn1 + idx;
        const uint32_t in2Offset = idy * rowSizeIn2 + idx;
        const uint32_t outOffset = idy * rowSizeOut + idx;

        // Store values locally to reduce global memory accesses
        const uint8_t valIn1 = in1[in1Offset];
        const uint8_t valIn2 = in2[in2Offset];

        // Calculate absolute difference and store the result
        out[outOffset] = abs(valIn1 - valIn2);
    }
}