#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flipCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height, bool horizontal, bool vertical)
{
    // Use shared memory to increase memory access speed
    __shared__ uint8_t tile[32][32 + 1]; // For potential bank conflict avoidance

    const uint32_t inX = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t inY = blockDim.y * blockIdx.y + threadIdx.y;

    if (inX < width && inY < height) {
        // Read input to shared memory
        tile[threadIdx.y][threadIdx.x] = in[inY * rowSizeIn + inX];
        __syncthreads();

        const uint32_t outX = horizontal ? (width - 1 - inX) : inX;
        const uint32_t outY = vertical ? (height - 1 - inY) : inY;

        // Write back from shared memory
        out[outY * rowSizeOut + outX] = tile[threadIdx.y][threadIdx.x];
    }
}