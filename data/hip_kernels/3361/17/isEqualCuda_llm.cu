#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void isEqualCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint32_t width, uint32_t height, uint32_t *isEqual) {
    // Calculate unique flattened index for each thread.
    const uint32_t idx = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;

    // Check if index is within bounds.
    if (idx < width * height) {
        const uint32_t y = idx / width;
        const uint32_t x = idx % width;

        // Compare elements directly using the index.
        const uint32_t partsEqual = static_cast<uint32_t>(in1[y * rowSizeIn1 + x] == in2[y * rowSizeIn2 + x]);
        atomicAnd(isEqual, partsEqual);  // Ensure memory safety during concurrent updates.
    }
}