#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize by using shared memory for faster data access
__global__ void copyCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        // Copy data using shared memory
        __shared__ uint8_t tile[BLOCK_SIZE][BLOCK_SIZE];
        tile[threadIdx.y][threadIdx.x] = in[y * rowSizeIn + x];
        __syncthreads();
        out[y * rowSizeOut + x] = tile[threadIdx.y][threadIdx.x];
    }
}