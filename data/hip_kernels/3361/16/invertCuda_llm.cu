#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void invertCuda(const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate global thread ID to merge 2D indexing into 1D for coalesced memory access
    const uint32_t index = blockIdx.y * blockDim.y * rowSizeOut + blockIdx.x * blockDim.x + threadIdx.y * rowSizeOut + threadIdx.x;

    if (index < width * height) {
        // Use single index computation to access linear memory
        out[index] = ~in[index];
    }
}