#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void isAnyEqualCuda(const uint8_t *image, uint8_t *value, size_t valueCount, uint32_t width, uint32_t height, uint32_t *differenceCount)
{
    // Calculate global thread index
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Boundary check
    if (x < width && y < height) {
        const uint32_t id = y * width + x;
        uint8_t pixelValue = image[id];

        // Use shared memory for values to reduce global memory access
        __shared__ uint8_t sharedValues[256]; // assuming a reasonable size limit, can adjust as needed
        int threadCount = blockDim.x * blockDim.y;  // Total threads in block
        for (uint32_t i = threadIdx.x * blockDim.y + threadIdx.y; i < valueCount; i += threadCount) {
            sharedValues[i] = value[i];
        }

        __syncthreads();

        // Check equality
        bool equal = false;
        for (uint32_t i = 0; i < valueCount; ++i) {
            if (pixelValue == sharedValues[i]) {
                equal = true;
                break;
            }
        }

        // Atomic increment if equal
        if (equal) {
            atomicAdd(differenceCount, 1);
        }
    }
}